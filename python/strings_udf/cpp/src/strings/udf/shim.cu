/*
 * Copyright (c) 2022, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <cudf/strings/udf/case.cuh>
#include <cudf/strings/udf/char_types.cuh>
#include <cudf/strings/udf/replace.cuh>
#include <cudf/strings/udf/search.cuh>
#include <cudf/strings/udf/starts_with.cuh>
#include <cudf/strings/udf/strip.cuh>
#include <cudf/strings/udf/udf_string.cuh>

using namespace cudf::strings::udf;

extern "C" __device__ int len(int* nb_retval, void const* str)
{
  auto sv    = reinterpret_cast<cudf::string_view const*>(str);
  *nb_retval = sv->length();
  return 0;
}

extern "C" __device__ int startswith(bool* nb_retval, void const* str, void const* substr)
{
  auto str_view    = reinterpret_cast<cudf::string_view const*>(str);
  auto substr_view = reinterpret_cast<cudf::string_view const*>(substr);

  *nb_retval = starts_with(*str_view, *substr_view);
  return 0;
}

extern "C" __device__ int endswith(bool* nb_retval, void const* str, void const* substr)
{
  auto str_view    = reinterpret_cast<cudf::string_view const*>(str);
  auto substr_view = reinterpret_cast<cudf::string_view const*>(substr);

  *nb_retval = ends_with(*str_view, *substr_view);
  return 0;
}

extern "C" __device__ int contains(bool* nb_retval, void const* str, void const* substr)
{
  auto str_view    = reinterpret_cast<cudf::string_view const*>(str);
  auto substr_view = reinterpret_cast<cudf::string_view const*>(substr);

  *nb_retval = (str_view->find(*substr_view) != cudf::string_view::npos);
  return 0;
}

extern "C" __device__ int find(int* nb_retval, void const* str, void const* substr)
{
  auto str_view    = reinterpret_cast<cudf::string_view const*>(str);
  auto substr_view = reinterpret_cast<cudf::string_view const*>(substr);

  *nb_retval = str_view->find(*substr_view);
  return 0;
}

extern "C" __device__ int rfind(int* nb_retval, void const* str, void const* substr)
{
  auto str_view    = reinterpret_cast<cudf::string_view const*>(str);
  auto substr_view = reinterpret_cast<cudf::string_view const*>(substr);

  *nb_retval = str_view->rfind(*substr_view);
  return 0;
}

extern "C" __device__ int eq(bool* nb_retval, void const* str, void const* rhs)
{
  auto str_view = reinterpret_cast<cudf::string_view const*>(str);
  auto rhs_view = reinterpret_cast<cudf::string_view const*>(rhs);

  *nb_retval = (*str_view == *rhs_view);
  return 0;
}

extern "C" __device__ int ne(bool* nb_retval, void const* str, void const* rhs)
{
  auto str_view = reinterpret_cast<cudf::string_view const*>(str);
  auto rhs_view = reinterpret_cast<cudf::string_view const*>(rhs);

  *nb_retval = (*str_view != *rhs_view);
  return 0;
}

extern "C" __device__ int ge(bool* nb_retval, void const* str, void const* rhs)
{
  auto str_view = reinterpret_cast<cudf::string_view const*>(str);
  auto rhs_view = reinterpret_cast<cudf::string_view const*>(rhs);

  *nb_retval = (*str_view >= *rhs_view);
  return 0;
}

extern "C" __device__ int le(bool* nb_retval, void const* str, void const* rhs)
{
  auto str_view = reinterpret_cast<cudf::string_view const*>(str);
  auto rhs_view = reinterpret_cast<cudf::string_view const*>(rhs);

  *nb_retval = (*str_view <= *rhs_view);
  return 0;
}

extern "C" __device__ int gt(bool* nb_retval, void const* str, void const* rhs)
{
  auto str_view = reinterpret_cast<cudf::string_view const*>(str);
  auto rhs_view = reinterpret_cast<cudf::string_view const*>(rhs);

  *nb_retval = (*str_view > *rhs_view);
  return 0;
}

extern "C" __device__ int lt(bool* nb_retval, void const* str, void const* rhs)
{
  auto str_view = reinterpret_cast<cudf::string_view const*>(str);
  auto rhs_view = reinterpret_cast<cudf::string_view const*>(rhs);

  *nb_retval = (*str_view < *rhs_view);
  return 0;
}

extern "C" __device__ int pyislower(bool* nb_retval, void const* str, std::uintptr_t chars_table)
{
  auto str_view = reinterpret_cast<cudf::string_view const*>(str);

  *nb_retval = is_lower(
    reinterpret_cast<cudf::strings::detail::character_flags_table_type*>(chars_table), *str_view);
  return 0;
}

extern "C" __device__ int pyisupper(bool* nb_retval, void const* str, std::uintptr_t chars_table)
{
  auto str_view = reinterpret_cast<cudf::string_view const*>(str);

  *nb_retval = is_upper(
    reinterpret_cast<cudf::strings::detail::character_flags_table_type*>(chars_table), *str_view);
  return 0;
}

extern "C" __device__ int pyisspace(bool* nb_retval, void const* str, std::uintptr_t chars_table)
{
  auto str_view = reinterpret_cast<cudf::string_view const*>(str);

  *nb_retval = is_space(
    reinterpret_cast<cudf::strings::detail::character_flags_table_type*>(chars_table), *str_view);
  return 0;
}

extern "C" __device__ int pyisdecimal(bool* nb_retval, void const* str, std::uintptr_t chars_table)
{
  auto str_view = reinterpret_cast<cudf::string_view const*>(str);

  *nb_retval = is_decimal(
    reinterpret_cast<cudf::strings::detail::character_flags_table_type*>(chars_table), *str_view);
  return 0;
}

extern "C" __device__ int pyisnumeric(bool* nb_retval, void const* str, std::uintptr_t chars_table)
{
  auto str_view = reinterpret_cast<cudf::string_view const*>(str);

  *nb_retval = is_numeric(
    reinterpret_cast<cudf::strings::detail::character_flags_table_type*>(chars_table), *str_view);
  return 0;
}

extern "C" __device__ int pyisdigit(bool* nb_retval, void const* str, std::uintptr_t chars_table)
{
  auto str_view = reinterpret_cast<cudf::string_view const*>(str);

  *nb_retval = is_digit(
    reinterpret_cast<cudf::strings::detail::character_flags_table_type*>(chars_table), *str_view);
  return 0;
}

extern "C" __device__ int pyisalnum(bool* nb_retval, void const* str, std::uintptr_t chars_table)
{
  auto str_view = reinterpret_cast<cudf::string_view const*>(str);

  *nb_retval = is_alpha_numeric(
    reinterpret_cast<cudf::strings::detail::character_flags_table_type*>(chars_table), *str_view);
  return 0;
}

extern "C" __device__ int pyisalpha(bool* nb_retval, void const* str, std::uintptr_t chars_table)
{
  auto str_view = reinterpret_cast<cudf::string_view const*>(str);

  *nb_retval = is_alpha(
    reinterpret_cast<cudf::strings::detail::character_flags_table_type*>(chars_table), *str_view);
  return 0;
}

extern "C" __device__ int pyistitle(bool* nb_retval, void const* str, std::uintptr_t chars_table)
{
  auto str_view = reinterpret_cast<cudf::string_view const*>(str);

  *nb_retval = is_title(
    reinterpret_cast<cudf::strings::detail::character_flags_table_type*>(chars_table), *str_view);
  return 0;
}

extern "C" __device__ int pycount(int* nb_retval, void const* str, void const* substr)
{
  auto str_view    = reinterpret_cast<cudf::string_view const*>(str);
  auto substr_view = reinterpret_cast<cudf::string_view const*>(substr);

  *nb_retval = count(*str_view, *substr_view);
  return 0;
}

extern "C" __device__ int udf_string_from_string_view(int* nb_retbal,
                                                      void const* str,
                                                      void* udf_str)
{
  auto str_view_ptr = reinterpret_cast<cudf::string_view const*>(str);
  auto udf_str_ptr  = new (udf_str) udf_string;
  *udf_str_ptr      = udf_string(*str_view_ptr);

  return 0;
}

extern "C" __device__ int strip(int* nb_retval,
                                void* udf_str,
                                void* const* to_strip,
                                void* const* strip_str)
{
  auto to_strip_ptr  = reinterpret_cast<cudf::string_view const*>(to_strip);
  auto strip_str_ptr = reinterpret_cast<cudf::string_view const*>(strip_str);
  auto udf_str_ptr   = new (udf_str) udf_string;

  *udf_str_ptr = strip(*to_strip_ptr, *strip_str_ptr);

  return 0;
}

extern "C" __device__ int lstrip(int* nb_retval,
                                 void* udf_str,
                                 void* const* to_strip,
                                 void* const* strip_str)
{
  auto to_strip_ptr  = reinterpret_cast<cudf::string_view const*>(to_strip);
  auto strip_str_ptr = reinterpret_cast<cudf::string_view const*>(strip_str);
  auto udf_str_ptr   = new (udf_str) udf_string;

  *udf_str_ptr = strip(*to_strip_ptr, *strip_str_ptr, cudf::strings::side_type::LEFT);

  return 0;
}

extern "C" __device__ int rstrip(int* nb_retval,
                                 void* udf_str,
                                 void* const* to_strip,
                                 void* const* strip_str)
{
  auto to_strip_ptr  = reinterpret_cast<cudf::string_view const*>(to_strip);
  auto strip_str_ptr = reinterpret_cast<cudf::string_view const*>(strip_str);
  auto udf_str_ptr   = new (udf_str) udf_string;

  *udf_str_ptr = strip(*to_strip_ptr, *strip_str_ptr, cudf::strings::side_type::RIGHT);

  return 0;
}
extern "C" __device__ int upper(int* nb_retval,
                                void* udf_str,
                                void const* st,
                                std::uintptr_t flags_table,
                                std::uintptr_t cases_table,
                                std::uintptr_t special_table)
{
  auto udf_str_ptr = new (udf_str) udf_string;
  auto st_ptr      = reinterpret_cast<cudf::string_view const*>(st);

  auto flags_table_ptr =
    reinterpret_cast<cudf::strings::detail::character_flags_table_type*>(flags_table);
  auto cases_table_ptr =
    reinterpret_cast<cudf::strings::detail::character_cases_table_type*>(cases_table);
  auto special_table_ptr =
    reinterpret_cast<cudf::strings::detail::special_case_mapping*>(special_table);

  cudf::strings::udf::chars_tables tables{flags_table_ptr, cases_table_ptr, special_table_ptr};

  *udf_str_ptr = to_upper(tables, *st_ptr);

  return 0;
}

extern "C" __device__ int lower(int* nb_retval,
                                void* udf_str,
                                void const* st,
                                std::uintptr_t flags_table,
                                std::uintptr_t cases_table,
                                std::uintptr_t special_table)
{
  auto udf_str_ptr = new (udf_str) udf_string;
  auto st_ptr      = reinterpret_cast<cudf::string_view const*>(st);

  auto flags_table_ptr =
    reinterpret_cast<cudf::strings::detail::character_flags_table_type*>(flags_table);
  auto cases_table_ptr =
    reinterpret_cast<cudf::strings::detail::character_cases_table_type*>(cases_table);
  auto special_table_ptr =
    reinterpret_cast<cudf::strings::detail::special_case_mapping*>(special_table);

  cudf::strings::udf::chars_tables tables{flags_table_ptr, cases_table_ptr, special_table_ptr};
  *udf_str_ptr = to_lower(tables, *st_ptr);
  return 0;
}

extern "C" __device__ int concat(int* nb_retval, void* udf_str, void* const* lhs, void* const* rhs)
{
  auto lhs_ptr = reinterpret_cast<cudf::string_view const*>(lhs);
  auto rhs_ptr = reinterpret_cast<cudf::string_view const*>(rhs);

  auto udf_str_ptr = new (udf_str) udf_string;

  udf_string result;
  result.append(*lhs_ptr).append(*rhs_ptr);
  *udf_str_ptr = result;
  return 0;
}

extern "C" __device__ int replace(
  int* nb_retval, void* udf_str, void* const src, void* const to_replace, void* const replacement)
{
  auto src_ptr         = reinterpret_cast<cudf::string_view const*>(src);
  auto to_replace_ptr  = reinterpret_cast<cudf::string_view const*>(to_replace);
  auto replacement_ptr = reinterpret_cast<cudf::string_view const*>(replacement);

  auto udf_str_ptr = new (udf_str) udf_string;
  *udf_str_ptr     = replace(*src_ptr, *to_replace_ptr, *replacement_ptr);

  return 0;
}
