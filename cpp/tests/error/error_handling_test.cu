#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2018, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */
#include <utilities/error_utils.hpp>
#include <cudf/cudf.h>

#include <rmm/rmm.h>

#include <tests/utilities/legacy/cudf_test_fixtures.h>

#include <cstring>

// If this test fails, it means an error code was added without
// adding support to gdf_error_get_name().
TEST(ErrorTest, NameEveryError) {
  for (int i = 0; i < N_GDF_ERRORS; i++) {
    const char* res = gdf_error_get_name((gdf_error)i);
    ASSERT_EQ(0, strstr(res, "Unknown error"));
  }
}

TEST(ExpectsTest, FalseCondition) {
  EXPECT_THROW(CUDF_EXPECTS(false, "condition is false"), cudf::logic_error);
}

TEST(ExpectsTest, TrueCondition) {
  EXPECT_NO_THROW(CUDF_EXPECTS(true, "condition is true"));
}

TEST(ExpectsTest, TryCatch) {
  CUDF_EXPECT_THROW_MESSAGE(CUDF_EXPECTS(false, "test reason"), 
                            "test reason");
}

TEST(CudaTryTest, Error) {
  CUDA_EXPECT_THROW_MESSAGE(CUDA_TRY(hipErrorLaunchFailure),
                            "hipErrorLaunchFailure unspecified launch failure");
}
TEST(CudaTryTest, Success) { EXPECT_NO_THROW(CUDA_TRY(hipSuccess)); }

TEST(CudaTryTest, TryCatch) {
  CUDA_EXPECT_THROW_MESSAGE(CUDA_TRY(hipErrorOutOfMemory),
                            "hipErrorOutOfMemory out of memory");
}

TEST(StreamCheck, success) {
  EXPECT_NO_THROW(cudf::detail::check_stream(0, __FILE__, __LINE__));
}

namespace {
// Some silly kernel that will cause an error
void __global__ test_kernel(int* data) { data[threadIdx.x] = threadIdx.x; }
}  // namespace

// Test the function underlying CHECK_STREAM so that it throws an exception when
// a kernel fails
TEST(StreamCheck, FailedKernel) {
  hipStream_t stream;
  hipStreamCreate(&stream);
  int a;
  test_kernel<<<0, 0, 0, stream>>>(&a);
  EXPECT_THROW(cudf::detail::check_stream(0, __FILE__, __LINE__),
               cudf::cuda_error);
  hipStreamDestroy(stream);
}

TEST(StreamCheck, CatchFailedKernel) {
  hipStream_t stream;
  hipStreamCreate(&stream);
  int a;
  test_kernel<<<0, 0, 0, stream>>>(&a);
  CUDA_EXPECT_THROW_MESSAGE(cudf::detail::check_stream(0, __FILE__, __LINE__),
                            "hipErrorInvalidConfiguration "
                            "invalid configuration argument");
  hipStreamDestroy(stream);
}

// CHECK_STREAM should do nothing in a release build, even if there's an error
#ifdef NDEBUG
TEST(StreamCheck, ReleaseFailedKernel) {
  hipStream_t stream;
  hipStreamCreate(&stream);
  int a;
  test_kernel<<<0, 0, 0, stream>>>(&a);
  EXPECT_NO_THROW(CHECK_STREAM(0));
  hipStreamDestroy(stream);
}
#endif

// STREAM_CHECK only works in a non-Release build
#ifndef NDEBUG
TEST(StreamCheck, test) { EXPECT_NO_THROW(CHECK_STREAM(0)); }
#endif
