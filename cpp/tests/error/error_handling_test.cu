#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2018, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */
#include <utilities/error_utils.hpp>
#include <cudf.h>

#include <rmm/rmm.h>

#include <gtest/gtest.h>

#include <cstring>

// If this test fails, it means an error code was added without
// adding support to gdf_error_get_name().
TEST(ErrorTest, NameEveryError) {
  for (int i = 0; i < N_GDF_ERRORS; i++) {
    const char* res = gdf_error_get_name((gdf_error)i);
    ASSERT_EQ(0, strstr(res, "Unknown error"));
  }
}

TEST(ExpectsTest, FalseCondition) {
  EXPECT_THROW(CUDF_EXPECTS(false, "condition is false"), cudf::logic_error);
}

TEST(ExpectsTest, TrueCondition) {
  EXPECT_NO_THROW(CUDF_EXPECTS(true, "condition is true"));
}

TEST(ExpectsTest, TryCatch) {
  try {
    CUDF_EXPECTS(false, "test reason");
  } catch (cudf::logic_error const& e) {
    EXPECT_NE(nullptr, e.what());
    std::string what(e.what());
    EXPECT_NE(std::string::npos, what.find("cuDF failure at:"));
    EXPECT_NE(std::string::npos, what.find("test reason"));
  }
}

TEST(CudaTryTest, Error) {
  EXPECT_THROW(CUDA_TRY(hipErrorLaunchFailure), cudf::cuda_error);
}
TEST(CudaTryTest, Success) { EXPECT_NO_THROW(CUDA_TRY(hipSuccess)); }

TEST(CudaTryTest, TryCatch) {
  try {
    CUDA_TRY(hipErrorOutOfMemory);
  } catch (cudf::cuda_error const& e) {
    ASSERT_NE(nullptr, e.what());
    std::string what(e.what());
    EXPECT_NE(std::string::npos, what.find("CUDA error encountered at"));
    EXPECT_NE(std::string::npos, what.find("hipErrorOutOfMemory"));
  }
}

TEST(StreamCheck, success) {
  EXPECT_NO_THROW(cudf::detail::check_stream(0, __FILE__, __LINE__));
}

namespace {
// Some silly kernel that will cause an error
void __global__ test_kernel(int* data) { data[threadIdx.x] = threadIdx.x; }
}  // namespace

// Test the function underlying CHECK_STREAM so that it throws an exception when
// a kernel fails
TEST(StreamCheck, FailedKernel) {
  hipStream_t stream;
  hipStreamCreate(&stream);
  int a;
  test_kernel<<<0, 0, 0, stream>>>(&a);
  EXPECT_THROW(cudf::detail::check_stream(0, __FILE__, __LINE__),
               cudf::cuda_error);
  hipStreamDestroy(stream);
}

TEST(StreamCheck, CatchFailedKernel) {
  hipStream_t stream;
  hipStreamCreate(&stream);
  int a;
  test_kernel<<<0, 0, 0, stream>>>(&a);
  try {
    cudf::detail::check_stream(0, __FILE__, __LINE__);
  } catch (cudf::cuda_error const& e) {
    ASSERT_NE(nullptr, e.what());
    std::string what(e.what());
    EXPECT_NE(std::string::npos, what.find("CUDA error encountered at"));
    EXPECT_NE(
        std::string::npos,
        what.find(
            "hipErrorInvalidConfiguration invalid configuration argument"));
  }
  hipStreamDestroy(stream);
}

// CHECK_STREAM should do nothing in a release build, even if there's an error
#ifdef NDEBUG
TEST(StreamCheck, ReleaseFailedKernel) {
  hipStream_t stream;
  hipStreamCreate(&stream);
  int a;
  test_kernel<<<0, 0, 0, stream>>>(&a);
  EXPECT_NO_THROW(CHECK_STREAM(0));
  hipStreamDestroy(stream);
}
#endif

// STREAM_CHECK only works in a non-Release build
#ifndef NDEBUG
TEST(StreamCheck, test) { EXPECT_NO_THROW(CHECK_STREAM(0)); }
#endif
