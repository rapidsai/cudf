#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2018-2022, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <cudf_test/base_fixture.hpp>

#include <cudf/filling.hpp>
#include <cudf/utilities/default_stream.hpp>
#include <cudf/utilities/error.hpp>

#include <rmm/hip_stream.hpp>

TEST(ExpectsTest, FalseCondition)
{
  EXPECT_THROW(CUDF_EXPECTS(false, "condition is false"), cudf::logic_error);
}

TEST(ExpectsTest, TrueCondition) { EXPECT_NO_THROW(CUDF_EXPECTS(true, "condition is true")); }

TEST(CudaTryTest, Error) { EXPECT_THROW(CUDF_CUDA_TRY(hipErrorLaunchFailure), cudf::cuda_error); }

TEST(CudaTryTest, Success) { EXPECT_NO_THROW(CUDF_CUDA_TRY(hipSuccess)); }

TEST(StreamCheck, success) { EXPECT_NO_THROW(CUDF_CHECK_CUDA(0)); }

namespace {
// Some silly kernel that will cause an error
void __global__ test_kernel(int* data) { data[threadIdx.x] = threadIdx.x; }
}  // namespace

// In a release build and without explicit synchronization, CUDF_CHECK_CUDA may
// or may not fail on erroneous asynchronous CUDA calls. Invoke
// hipStreamSynchronize to guarantee failure on error. In a non-release build,
// CUDF_CHECK_CUDA deterministically fails on erroneous asynchronous CUDA
// calls.
TEST(StreamCheck, FailedKernel)
{
  rmm::hip_stream stream;
  int a;
  test_kernel<<<0, 0, 0, stream.value()>>>(&a);
#ifdef NDEBUG
  stream.synchronize();
#endif
  EXPECT_THROW(CUDF_CHECK_CUDA(stream.value()), cudf::cuda_error);
}

TEST(StreamCheck, CatchFailedKernel)
{
  rmm::hip_stream stream;
  int a;
  test_kernel<<<0, 0, 0, stream.value()>>>(&a);
#ifndef NDEBUG
  stream.synchronize();
#endif
  EXPECT_THROW(CUDF_CHECK_CUDA(stream.value()), cudf::cuda_error);
}

__global__ void kernel() { asm("trap;"); }

TEST(DeathTest, CudaFatalError)
{
  testing::FLAGS_gtest_death_test_style = "threadsafe";
  auto call_kernel                      = []() {
    kernel<<<1, 1, 0, cudf::get_default_stream().value()>>>();
    try {
      CUDF_CUDA_TRY(hipDeviceSynchronize());
    } catch (const cudf::fatal_cuda_error& fe) {
      std::abort();
    }
  };
  ASSERT_DEATH(call_kernel(), "");
}

#ifndef NDEBUG

__global__ void assert_false_kernel() { cudf_assert(false && "this kernel should die"); }

__global__ void assert_true_kernel() { cudf_assert(true && "this kernel should live"); }

TEST(DebugAssertDeathTest, cudf_assert_false)
{
  testing::FLAGS_gtest_death_test_style = "threadsafe";

  auto call_kernel = []() {
    assert_false_kernel<<<1, 1>>>();

    // Kernel should fail with `hipErrorAssert`
    // This error invalidates the current device context, so we need to kill
    // the current process. Running with EXPECT_DEATH spawns a new process for
    // each attempted kernel launch
    if (hipErrorAssert == hipDeviceSynchronize()) { std::abort(); }

    // If we reach this point, the cudf_assert didn't work so we exit normally, which will cause
    // EXPECT_DEATH to fail.
  };

  EXPECT_DEATH(call_kernel(), "this kernel should die");
}

TEST(DebugAssert, cudf_assert_true)
{
  assert_true_kernel<<<1, 1>>>();
  ASSERT_EQ(hipSuccess, hipDeviceSynchronize());
}

#endif

// These tests don't use CUDF_TEST_PROGRAM_MAIN because :
// 1.) They don't need the RMM Pool
// 2.) The RMM Pool interferes with the death test
int main(int argc, char** argv)
{
  ::testing::InitGoogleTest(&argc, argv);
  auto const cmd_opts    = parse_cudf_test_opts(argc, argv);
  auto const stream_mode = cmd_opts["stream_mode"].as<std::string>();
  if (stream_mode == "custom") {
    auto resource = rmm::mr::get_current_device_resource();
    auto adapter  = make_stream_checking_resource_adaptor(resource);
    rmm::mr::set_current_device_resource(&adapter);
  }
  return RUN_ALL_TESTS();
}
