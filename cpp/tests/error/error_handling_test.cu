#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2018, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */
#include <cudf/utilities/error.hpp>

#include <cudf_test/base_fixture.hpp>

#include <cstring>

TEST(ExpectsTest, FalseCondition)
{
  EXPECT_THROW(CUDF_EXPECTS(false, "condition is false"), cudf::logic_error);
}

TEST(ExpectsTest, TrueCondition) { EXPECT_NO_THROW(CUDF_EXPECTS(true, "condition is true")); }

TEST(ExpectsTest, TryCatch)
{
  CUDF_EXPECT_THROW_MESSAGE(CUDF_EXPECTS(false, "test reason"), "test reason");
}

TEST(CudaTryTest, Error)
{
  CUDA_EXPECT_THROW_MESSAGE(CUDA_TRY(hipErrorLaunchFailure),
                            "hipErrorLaunchFailure unspecified launch failure");
}
TEST(CudaTryTest, Success) { EXPECT_NO_THROW(CUDA_TRY(hipSuccess)); }

TEST(CudaTryTest, TryCatch)
{
  CUDA_EXPECT_THROW_MESSAGE(CUDA_TRY(hipErrorOutOfMemory),
                            "hipErrorOutOfMemory out of memory");
}

TEST(StreamCheck, success) { EXPECT_NO_THROW(CHECK_CUDA(0)); }

namespace {
// Some silly kernel that will cause an error
void __global__ test_kernel(int* data) { data[threadIdx.x] = threadIdx.x; }
}  // namespace

// In a release build and without explicit synchronization, CHECK_CUDA may
// or may not fail on erroneous asynchronous CUDA calls. Invoke
// hipStreamSynchronize to guarantee failure on error. In a non-release build,
// CHECK_CUDA deterministically fails on erroneous asynchronous CUDA
// calls.
TEST(StreamCheck, FailedKernel)
{
  hipStream_t stream;
  CUDA_TRY(hipStreamCreate(&stream));
  int a;
  test_kernel<<<0, 0, 0, stream>>>(&a);
#ifdef NDEBUG
  CUDA_TRY(hipStreamSynchronize(stream));
#endif
  EXPECT_THROW(CHECK_CUDA(stream), cudf::cuda_error);
  CUDA_TRY(hipStreamDestroy(stream));
}

TEST(StreamCheck, CatchFailedKernel)
{
  hipStream_t stream;
  CUDA_TRY(hipStreamCreate(&stream));
  int a;
  test_kernel<<<0, 0, 0, stream>>>(&a);
#ifndef NDEBUG
  CUDA_TRY(hipStreamSynchronize(stream));
#endif
  CUDA_EXPECT_THROW_MESSAGE(CHECK_CUDA(stream),
                            "hipErrorInvalidConfiguration "
                            "invalid configuration argument");
  CUDA_TRY(hipStreamDestroy(stream));
}

CUDF_TEST_PROGRAM_MAIN()
