/*
 * Copyright (c) 2019, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <cudf/filling.hpp>
#include <tests/utilities/column_wrapper.cuh>
#include <tests/utilities/scalar_wrapper.cuh>
#include <tests/utilities/cudf_test_fixtures.h>

template <typename T>
using column_wrapper = cudf::test::column_wrapper<T>;

template <typename T>
using scalar_wrapper = cudf::test::scalar_wrapper<T>;

template <typename T>
struct FillingTest : GdfTest {};

using test_types =
  ::testing::Types<int8_t, int16_t, int32_t, int64_t, float, double,
                   cudf::bool8>;
TYPED_TEST_CASE(FillingTest, test_types);

constexpr gdf_size_type column_size{1000};

auto all_valid = [](gdf_index_type row) { return true; };

template <typename T, typename BitInitializerType = decltype(all_valid)>
void FillTest(gdf_index_type begin, gdf_index_type end,
              T value, bool value_is_valid = true, 
              BitInitializerType source_validity = all_valid)
{
  column_wrapper<T> source(column_size, 
    [](gdf_index_type row) { return static_cast<T>(row); },
    [&](gdf_index_type row) { return source_validity(row); });

  scalar_wrapper<T> val(value, value_is_valid);

  column_wrapper<T> expected(column_size,
    [&](gdf_index_type row) { 
      return (row >= begin && row < end) ? 
        value : static_cast<T>(row);
    },
    [&](gdf_index_type row) { 
      return (row >= begin && row < end) ? 
        value_is_valid : source_validity(row); 
    });

  EXPECT_NO_THROW(cudf::fill(source.get(), *val.get(), begin, end));

  EXPECT_TRUE(expected == source);

  if (!(expected == source)) {
    std::cout << "expected\n";
    expected.print();
    std::cout << expected.get()->null_count << "\n";
    std::cout << "source\n";
    source.print();
    std::cout << source.get()->null_count << "\n";
  }
}

TYPED_TEST(FillingTest, SetSingle)
{
  gdf_index_type index = 9;
  TypeParam val = TypeParam{1};
  
  // First set it as valid
  FillTest(index, index+1, val, true);
  // next set it as invalid
  FillTest(index, index+1, val, false);
}

TYPED_TEST(FillingTest, SetAll)
{
  TypeParam val = TypeParam{1};

  // First set it as valid
  FillTest(0, column_size, val, true);
  // next set it as invalid
  FillTest(0, column_size, val, false);
}

TYPED_TEST(FillingTest, SetRange)
{
  gdf_index_type begin = 99;
  gdf_index_type end   = 299;
  TypeParam val = TypeParam{1};

  // First set it as valid
  FillTest(begin, end, val, true);
  // Next set it as invalid
  FillTest(begin, end, val, false);
}

TYPED_TEST(FillingTest, SetRangeNullCount)
{
  gdf_index_type begin = 10;
  gdf_index_type end = 50;
  TypeParam val = TypeParam{1};

  auto some_valid = [](gdf_index_type row) { 
    return row % 2 != 0;
  };

  auto all_invalid = [](gdf_index_type row) { 
    return false;
  };

  // First set it as valid value
  FillTest(begin, end, val, true, some_valid);

  // Next set it as invalid
  FillTest(begin, end, val, false, some_valid);

  // All invalid column should have some valid
  FillTest(begin, end, val, true, all_invalid);

  // All should be invalid
  FillTest(begin, end, val, false, all_invalid);

  // All should be valid
  FillTest(0, column_size, val, true, some_valid);
}

struct FillingErrorTest : GdfTest {};

TEST_F(FillingErrorTest, InvalidColumn)
{
  scalar_wrapper<int32_t> val(5, true);
  CUDF_EXPECT_THROW_MESSAGE(cudf::fill(nullptr, *val.get(), 0, 10),
                            "Null gdf_column pointer");

  gdf_column bad_input;
  gdf_column_view(&bad_input, 0, 0, 0, GDF_INT32);
  // empty range == no-op, even on invalid output column...
  EXPECT_NO_THROW(cudf::fill(&bad_input, *val.get(), 0, 0));

  // for zero-size column, non-empty range is out of bounds
  CUDF_EXPECT_THROW_MESSAGE(cudf::fill(&bad_input, *val.get(), 0, 10),
                            "Range is out of bounds");

  // invalid data pointer
  bad_input.size = 20;
  CUDF_EXPECT_THROW_MESSAGE(cudf::fill(&bad_input, *val.get(), 0, 10),
                            "Null column data with non-zero size");
}

TEST_F(FillingErrorTest, InvalidRange)
{
  scalar_wrapper<int32_t> val(5, true);
  column_wrapper<int32_t> dest(100, 
    [](gdf_index_type row) { return static_cast<int32_t>(row); },
    [](gdf_index_type row) { return true; });
  
  CUDF_EXPECT_THROW_MESSAGE(cudf::fill(dest.get(), *val.get(), 0, 110),
                            "Range is out of bounds");
  CUDF_EXPECT_THROW_MESSAGE(cudf::fill(dest.get(), *val.get(), -10, 0),
                            "Range is out of bounds");
  CUDF_EXPECT_THROW_MESSAGE(cudf::fill(dest.get(), *val.get(), 10, 0),
                            "Range is empty or reversed");
}

TEST_F(FillingErrorTest, DTypeMismatch)
{
  scalar_wrapper<int32_t> val(5, true);
  column_wrapper<float> dest(100, 
    [](gdf_index_type row) { return static_cast<float>(row); },
    [](gdf_index_type row) { return true; });
  CUDF_EXPECT_THROW_MESSAGE(cudf::fill(dest.get(), *val.get(), 0, 10),
                            "Data type mismatch");
}

TEST_F(FillingErrorTest, StringCategoryNotSupported)
{
  scalar_wrapper<int32_t> val(5, true);
  std::vector<const char*> strings{"foo"};
  column_wrapper<cudf::nvstring_category> dest(1, strings.data());
  CUDF_EXPECT_THROW_MESSAGE(cudf::fill(dest.get(), *val.get(), 0, 1),
    "cudf::fill() does not support GDF_STRING_CATEGORY columns");
}