/*
 * Copyright 2018 BlazingDB, Inc.
 *     Copyright 2018 Alexander Ocsa <cristhian@blazingdb.com>
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <cudf/replace.hpp>

#include <utilities/error_utils.hpp>

#include <tests/utilities/legacy/column_wrapper.cuh>
#include <tests/utilities/legacy/scalar_wrapper.cuh>
#include <tests/utilities/legacy/cudf_test_fixtures.h>
#include <tests/utilities/legacy/cudf_test_utils.cuh>
#include <cudf/cudf.h>


template <typename T>
struct ReplaceNullsTest : GdfTest {};

using test_types =
    ::testing::Types<int8_t, int16_t, int32_t, int64_t, float, double>;

TYPED_TEST_CASE(ReplaceNullsTest, test_types);

template <typename T>
void ReplaceNullsColumn(cudf::test::column_wrapper<T> input,
                        cudf::test::column_wrapper<T> replacement_values,
                        cudf::test::column_wrapper<T> expected)
{
  gdf_column result;
  EXPECT_NO_THROW(result = cudf::replace_nulls(input, replacement_values));
  EXPECT_TRUE(expected == result);
}

template <typename T>
void ReplaceNullsScalar(cudf::test::column_wrapper<T> input,
                        cudf::test::scalar_wrapper<T> replacement_value,
                        cudf::test::column_wrapper<T> expected)
{
  gdf_column result;
  EXPECT_NO_THROW(result = cudf::replace_nulls(input, replacement_value));
  EXPECT_TRUE(expected == result);
}

TYPED_TEST(ReplaceNullsTest, ReplaceColumn)
{
  constexpr gdf_size_type column_size{10};

  ReplaceNullsColumn<TypeParam>(
    cudf::test::column_wrapper<TypeParam> {column_size,
      [](gdf_index_type row) { return row; },
      [](gdf_index_type row) { return (row < column_size/2) ? false : true; }},
    cudf::test::column_wrapper<TypeParam> {column_size,
      [](gdf_index_type row) { return 1; },
      false},
    cudf::test::column_wrapper<TypeParam> {column_size,
      [](gdf_index_type row) { return (row < column_size/2) ? 1 : row; },
      false});
}


TYPED_TEST(ReplaceNullsTest, ReplaceScalar)
{
  constexpr gdf_size_type column_size{10};

  ReplaceNullsScalar<TypeParam>(
    cudf::test::column_wrapper<TypeParam> {column_size,
      [](gdf_index_type row) { return row; },
      [](gdf_index_type row) { return (row < column_size/2) ? false : true; }},
    cudf::test::scalar_wrapper<TypeParam> {1, true},
    cudf::test::column_wrapper<TypeParam> {column_size,
      [](gdf_index_type row) { return (row < column_size/2) ? 1 : row; },
      false});
}