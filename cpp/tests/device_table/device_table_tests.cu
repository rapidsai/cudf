/*
 * Copyright (c) 2019, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <bitmask/legacy/bit_mask.cuh>
#include <bitmask/legacy/bitmask_ops.hpp>
#include <table/device_table.cuh>
#include <table/device_table_row_operators.cuh>
#include <gmock/gmock.h>
#include <gtest/gtest.h>
#include <tests/utilities/column_wrapper.cuh>
#include <tests/utilities/cudf_test_fixtures.h>
#include <tests/utilities/cudf_test_utils.cuh>
#include <cudf/types.hpp>

#include <thrust/execution_policy.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/logical.h>

#include <numeric>
#include <random>

struct DeviceTableTest : GdfTest {
  gdf_size_type const size{2000};
};

/**---------------------------------------------------------------------------*
 * @brief Compares if a row in one table is equal to all rows in another table.
 *
 *---------------------------------------------------------------------------**/
template <bool nullable>
struct all_rows_equal {
  device_table lhs;
  device_table rhs;
  bool nulls_are_equal;

  all_rows_equal(device_table _lhs, device_table _rhs,
                 bool _nulls_are_equal = false)
      : lhs{_lhs}, rhs{_rhs}, nulls_are_equal{_nulls_are_equal} {}

  /**---------------------------------------------------------------------------*
   * @brief Returns true if row `lhs_index` in the `lhs` table is equal to every
   * row in the `rhs` table.
   *
   *---------------------------------------------------------------------------**/
  __device__ bool operator()(int lhs_index) {
    auto row_equality = [this, lhs_index](gdf_size_type rhs_index) {
      return rows_equal<nullable>(lhs, lhs_index, rhs, rhs_index,
                                  nulls_are_equal);
    };
    return thrust::all_of(thrust::seq, thrust::make_counting_iterator(0),
                          thrust::make_counting_iterator(rhs.num_rows()),
                          row_equality);
  }
};

template <bool nullable>
struct row_comparison {
  row_equality_comparator<nullable> comp;

  using index_pair = thrust::tuple<gdf_size_type, gdf_size_type>;

  row_comparison(device_table _lhs, device_table _rhs,
                 bool _nulls_are_equal = false)
      : comp{_lhs, _rhs, _nulls_are_equal} {}

  __device__ bool operator()(index_pair const& indices) {
    return comp(thrust::get<0>(indices), thrust::get<1>(indices));
  }
};

template <bool nullable>
struct row_hasher {
  device_table t;
  row_hasher(device_table _t) : t{_t} {}
  __device__ hash_value_type operator()(gdf_size_type row_index) {
    return hash_row<nullable>(t, row_index);
  }
};

TEST_F(DeviceTableTest, AllRowsEqualNoNulls) {
  const int val{42};
  auto init_values = [val](auto index) { return val; };
  auto all_valid = [](auto index) { return true; };

  // 4 columns will all rows equal, no nulls
  cudf::test::column_wrapper<int32_t> col0(size, init_values, all_valid);
  cudf::test::column_wrapper<float> col1(size, init_values, all_valid);
  cudf::test::column_wrapper<double> col2(size, init_values, all_valid);
  cudf::test::column_wrapper<int8_t> col3(size, init_values, all_valid);

  std::vector<gdf_column*> gdf_cols{col0, col1, col2, col3};

  auto table = device_table::create(gdf_cols.size(), gdf_cols.data());

  // Every row should be equal to every other row regardless of NULL ?= NULL
  EXPECT_TRUE(thrust::all_of(rmm::exec_policy()->on(0),
                             thrust::make_counting_iterator(0),
                             thrust::make_counting_iterator(size),
                             all_rows_equal<false>(*table, *table, true)));
  EXPECT_TRUE(thrust::all_of(rmm::exec_policy()->on(0),
                             thrust::make_counting_iterator(0),
                             thrust::make_counting_iterator(size),
                             all_rows_equal<false>(*table, *table, false)));

  // Compute hash value of every row
  thrust::device_vector<hash_value_type> row_hashes(table->num_rows());
  thrust::tabulate(row_hashes.begin(), row_hashes.end(),
                   row_hasher<false>{*table});

  // All hash values should be equal
  EXPECT_TRUE(thrust::equal(row_hashes.begin() + 1, row_hashes.end(),
                            row_hashes.begin()));
}

TEST_F(DeviceTableTest, AllRowsEqualWithNulls) {
  const int val{42};
  auto init_values = [val](auto index) { return val; };
  auto all_valid = [](auto index) { return true; };
  auto all_null = [](auto index) { return false; };

  // 4 columns with all rows equal, last column is all nulls
  cudf::test::column_wrapper<int32_t> col0(size, init_values, all_valid);
  cudf::test::column_wrapper<float> col1(size, init_values, all_valid);
  cudf::test::column_wrapper<double> col2(size, init_values, all_valid);
  cudf::test::column_wrapper<int8_t> col3(size, init_values, all_null);

  std::vector<gdf_column*> gdf_cols{col0, col1, col2, col3};

  auto table = device_table::create(gdf_cols.size(), gdf_cols.data());

  // If NULL != NULL, no row can equal any other row
  EXPECT_FALSE(thrust::all_of(rmm::exec_policy()->on(0),
                              thrust::make_counting_iterator(0),
                              thrust::make_counting_iterator(size),
                              all_rows_equal<true>(*table, *table, false)));

  // If NULL == NULL, all rows should be equal
  EXPECT_TRUE(thrust::all_of(rmm::exec_policy()->on(0),
                             thrust::make_counting_iterator(0),
                             thrust::make_counting_iterator(size),
                             all_rows_equal<true>(*table, *table, true)));

  // Compute hash value of every row
  thrust::device_vector<hash_value_type> row_hashes(table->num_rows());
  thrust::tabulate(row_hashes.begin(), row_hashes.end(),
                   row_hasher<true>{*table});

  // All hash values should be equal because hash_row should ignore nulls
  EXPECT_TRUE(thrust::equal(row_hashes.begin() + 1, row_hashes.end(),
                            row_hashes.begin()));
}

TEST_F(DeviceTableTest, AllRowsDifferentWithNulls) {
  int const val{42};
  auto init_values = [val](auto index) { return index; };
  auto all_valid = [](auto index) { return true; };
  auto all_null = [](auto index) { return false; };

  // 4 columns with all rows different, last column is all nulls
  cudf::test::column_wrapper<int32_t> col0(size, init_values, all_valid);
  cudf::test::column_wrapper<float> col1(size, init_values, all_valid);
  cudf::test::column_wrapper<double> col2(size, init_values, all_valid);
  cudf::test::column_wrapper<int8_t> col3(size, init_values, all_null);

  std::vector<gdf_column*> gdf_cols{col0, col1, col2, col3};

  auto table = device_table::create(gdf_cols.size(), gdf_cols.data());

  // If NULL==NULL, every row should be equal to itself
  thrust::device_vector<gdf_size_type> indices(table->num_rows());
  thrust::sequence(indices.begin(), indices.end());
  EXPECT_TRUE(thrust::all_of(rmm::exec_policy()->on(0),
                             thrust::make_zip_iterator(thrust::make_tuple(
                                 indices.begin(), indices.begin())),
                             thrust::make_zip_iterator(thrust::make_tuple(
                                 indices.end(), indices.end())),
                             row_comparison<true>{*table, *table, true}));

  // If NULL!=NULL, every row should *not* be equal to itself
  EXPECT_FALSE(thrust::all_of(rmm::exec_policy()->on(0),
                              thrust::make_zip_iterator(thrust::make_tuple(
                                  indices.begin(), indices.begin())),
                              thrust::make_zip_iterator(thrust::make_tuple(
                                  indices.end(), indices.end())),
                              row_comparison<true>{*table, *table, false}));

  // Compute hash value of every row
  thrust::device_vector<hash_value_type> row_hashes(table->num_rows());
  thrust::tabulate(row_hashes.begin(), row_hashes.end(),
                   row_hasher<true>{*table});

  // All hash values should be NOT be equal
  EXPECT_FALSE(thrust::equal(row_hashes.begin() + 1, row_hashes.end(),
                             row_hashes.begin()));

  // Every row should be different from every other row other than itself
  for (gdf_size_type i = 0; i < table->num_rows(); ++i) {
    thrust::device_vector<gdf_size_type> left_indices(table->num_rows(), i);
    thrust::device_vector<gdf_size_type> right_indices(table->num_rows());
    thrust::sequence(right_indices.begin(), right_indices.end());

    // Remove indices comparing a row against itself
    left_indices.erase(left_indices.begin() + i);
    right_indices.erase(right_indices.begin() + i);

    // Ensure row `i` is not equal to every other row `j`, `i != j`
    EXPECT_FALSE(thrust::all_of(
        rmm::exec_policy()->on(0),
        thrust::make_zip_iterator(
            thrust::make_tuple(left_indices.begin(), right_indices.begin())),
        thrust::make_zip_iterator(
            thrust::make_tuple(left_indices.end(), right_indices.end())),
        row_comparison<true>{*table, *table, true}));
  }
}

TEST_F(DeviceTableTest, TwoTablesAllRowsEqual) {
  int const val{42};
  auto init_values = [val](auto index) { return index; };
  auto random_values = [](auto index) {
    return std::default_random_engine{}();
  };
  auto all_valid = [](auto index) { return true; };
  auto all_null = [](auto index) { return false; };

  cudf::test::column_wrapper<int32_t> left_col0(size, init_values, all_valid);
  cudf::test::column_wrapper<float> left_col1(size, init_values, all_valid);
  cudf::test::column_wrapper<double> left_col2(size, init_values, all_valid);
  cudf::test::column_wrapper<int8_t> left_col3(size, random_values, all_null);
  std::vector<gdf_column*> left_cols{left_col0, left_col1, left_col2,
                                     left_col3};
  auto left_table = device_table::create(left_cols.size(), left_cols.data());

  cudf::test::column_wrapper<int32_t> right_col0(size, init_values, all_valid);
  cudf::test::column_wrapper<float> right_col1(size, init_values, all_valid);
  cudf::test::column_wrapper<double> right_col2(size, init_values, all_valid);
  cudf::test::column_wrapper<int8_t> right_col3(size, random_values, all_null);
  std::vector<gdf_column*> right_cols{right_col0, right_col1, right_col2,
                                      right_col3};
  auto right_table = device_table::create(right_cols.size(), right_cols.data());

  // If NULL==NULL, left_table row @ i should equal right_table row @ i
  thrust::device_vector<gdf_size_type> indices(left_table->num_rows());
  thrust::sequence(indices.begin(), indices.end());
  EXPECT_TRUE(
      thrust::all_of(rmm::exec_policy()->on(0),
                     thrust::make_zip_iterator(
                         thrust::make_tuple(indices.begin(), indices.begin())),
                     thrust::make_zip_iterator(
                         thrust::make_tuple(indices.end(), indices.end())),
                     row_comparison<true>{*left_table, *right_table, true}));

  // If NULL!=NULL, left_table row @ i should NOT equal right_table row @ i
  EXPECT_FALSE(
      thrust::all_of(rmm::exec_policy()->on(0),
                     thrust::make_zip_iterator(
                         thrust::make_tuple(indices.begin(), indices.begin())),
                     thrust::make_zip_iterator(
                         thrust::make_tuple(indices.end(), indices.end())),
                     row_comparison<true>{*left_table, *right_table, false}));
}

template <bool update_target_bitmask>
struct row_copier {
  device_table target;
  device_table source;

  using index_pair = thrust::tuple<gdf_size_type, gdf_size_type>;

  row_copier(device_table _target, device_table _source)
      : target{_target}, source{_source} {}

  __device__ void operator()(index_pair const& indices) {
    copy_row<update_target_bitmask>(target, thrust::get<0>(indices), source,
                                    thrust::get<1>(indices));
  }
};

TEST_F(DeviceTableTest, CopyRowsNoNulls) {
  int const val{42};
  auto init_values = [val](auto index) { return index; };
  auto all_valid = [](auto index) { return true; };

  cudf::test::column_wrapper<int32_t> source_col0(size, init_values, all_valid);
  cudf::test::column_wrapper<float> source_col1(size, init_values, all_valid);
  cudf::test::column_wrapper<double> source_col2(size, init_values, all_valid);
  cudf::test::column_wrapper<int8_t> source_col3(size, init_values, all_valid);
  std::vector<gdf_column*> source_cols{source_col0, source_col1, source_col2,
                                       source_col3};
  auto source_table =
      device_table::create(source_cols.size(), source_cols.data());

  cudf::test::column_wrapper<int32_t> target_col0(size);
  cudf::test::column_wrapper<float> target_col1(size);
  cudf::test::column_wrapper<double> target_col2(size);
  cudf::test::column_wrapper<int8_t> target_col3(size);
  std::vector<gdf_column*> target_cols{target_col0, target_col1, target_col2,
                                       target_col3};
  auto target_table =
      device_table::create(target_cols.size(), target_cols.data());

  // Copy a random row from the source table to a random row in the target table
  // Thrust doesn't have a `shuffle` algorithm, so we've got to do it on the
  // host
  std::vector<gdf_size_type> indices(source_table->num_rows());
  std::iota(indices.begin(), indices.end(), 0);
  std::shuffle(indices.begin(), indices.end(), std::default_random_engine{});
  thrust::device_vector<gdf_size_type> target_indices(indices);

  std::shuffle(indices.begin(), indices.end(), std::default_random_engine{});
  thrust::device_vector<gdf_size_type> source_indices(indices);

  // Copy source_table row @ source_indices[i] to target_table @
  // target_indices[i]
  EXPECT_NO_THROW(thrust::for_each(
      rmm::exec_policy()->on(0),
      thrust::make_zip_iterator(
          thrust::make_tuple(target_indices.begin(), source_indices.begin())),
      thrust::make_zip_iterator(
          thrust::make_tuple(target_indices.end(), source_indices.end())),
      row_copier<false>{*target_table, *source_table}));

  // ensure source_table row @ source_indices[i] == target_table row @
  // target_indices[i]
  EXPECT_TRUE(thrust::all_of(
      rmm::exec_policy()->on(0),
      thrust::make_zip_iterator(
          thrust::make_tuple(source_indices.begin(), target_indices.begin())),
      thrust::make_zip_iterator(
          thrust::make_tuple(source_indices.end(), target_indices.end())),
      row_comparison<false>{*source_table, *target_table}));
}

struct verify_bitmask {
  bit_mask::bit_mask_t* bitmask;

  verify_bitmask(bit_mask::bit_mask_t* _bitmask) : bitmask{_bitmask} {}

  __device__ bool operator()(gdf_size_type i) {
    return bit_mask::is_valid(bitmask, i);
  }
};

TEST_F(DeviceTableTest, CopyRowsSourceNullTargetValid) {
  int const val{42};
  auto init_values = [val](auto index) { return index; };
  auto all_valid = [](auto index) { return true; };
  auto all_null = [](auto index) { return false; };

  cudf::test::column_wrapper<int32_t> source_col0(size, init_values, all_null);
  cudf::test::column_wrapper<float> source_col1(size, init_values, all_null);
  cudf::test::column_wrapper<double> source_col2(size, init_values, all_null);
  cudf::test::column_wrapper<int8_t> source_col3(size, init_values, all_null);
  std::vector<gdf_column*> source_cols{source_col0, source_col1, source_col2,
                                       source_col3};
  auto source_table =
      device_table::create(source_cols.size(), source_cols.data());

  cudf::test::column_wrapper<int32_t> target_col0(size, init_values, all_valid);
  cudf::test::column_wrapper<float> target_col1(size, init_values, all_valid);
  cudf::test::column_wrapper<double> target_col2(size, init_values, all_valid);
  cudf::test::column_wrapper<int8_t> target_col3(size, init_values, all_valid);
  std::vector<gdf_column*> target_cols{target_col0, target_col1, target_col2,
                                       target_col3};
  auto target_table =
      device_table::create(target_cols.size(), target_cols.data());

  // Copy a random row from the source table to a random row in the target table
  // Thrust doesn't have a `shuffle` algorithm, so we've got to do it on the
  // host
  std::vector<gdf_size_type> indices(source_table->num_rows());
  std::iota(indices.begin(), indices.end(), 0);
  std::shuffle(indices.begin(), indices.end(), std::default_random_engine{});
  thrust::device_vector<gdf_size_type> target_indices(indices);

  std::shuffle(indices.begin(), indices.end(), std::default_random_engine{});
  thrust::device_vector<gdf_size_type> source_indices(indices);

  // Copy source_table row @ source_indices[i] to target_table @
  // target_indices[i]
  EXPECT_NO_THROW(thrust::for_each(
      rmm::exec_policy()->on(0),
      thrust::make_zip_iterator(
          thrust::make_tuple(target_indices.begin(), source_indices.begin())),
      thrust::make_zip_iterator(
          thrust::make_tuple(target_indices.end(), source_indices.end())),
      row_copier<true>{*target_table, *source_table}));

  // Every source and target row should be all nulls

  // Rows should be equal if NULL == NULL
  EXPECT_TRUE(thrust::all_of(
      rmm::exec_policy()->on(0),
      thrust::make_zip_iterator(
          thrust::make_tuple(source_indices.begin(), target_indices.begin())),
      thrust::make_zip_iterator(
          thrust::make_tuple(source_indices.end(), target_indices.end())),
      row_comparison<true>{*source_table, *target_table, true}));

  // No row should be equal if NULL != NULL
  EXPECT_TRUE(thrust::none_of(
      rmm::exec_policy()->on(0),
      thrust::make_zip_iterator(
          thrust::make_tuple(source_indices.begin(), target_indices.begin())),
      thrust::make_zip_iterator(
          thrust::make_tuple(source_indices.end(), target_indices.end())),
      row_comparison<true>{*source_table, *target_table, false}));
}

TEST_F(DeviceTableTest, CopyRowsSourceValidTargetNull) {
  int const val{42};
  auto init_values = [val](auto index) { return index; };
  auto all_valid = [](auto index) { return true; };

  cudf::test::column_wrapper<int32_t> source_col0(size, init_values, all_valid);
  cudf::test::column_wrapper<float> source_col1(size, init_values, all_valid);
  cudf::test::column_wrapper<double> source_col2(size, init_values, all_valid);
  cudf::test::column_wrapper<int8_t> source_col3(size, init_values, all_valid);
  std::vector<gdf_column*> source_cols{source_col0, source_col1, source_col2,
                                       source_col3};
  auto source_table =
      device_table::create(source_cols.size(), source_cols.data());

  cudf::test::column_wrapper<int32_t> target_col0(size, true);
  cudf::test::column_wrapper<float> target_col1(size, true);
  cudf::test::column_wrapper<double> target_col2(size, true);
  cudf::test::column_wrapper<int8_t> target_col3(size, true);
  std::vector<gdf_column*> target_cols{target_col0, target_col1, target_col2,
                                       target_col3};

  auto target_table =
      device_table::create(target_cols.size(), target_cols.data());

  // Copy a random row from the source table to a random row in the target table
  // Thrust doesn't have a `shuffle` algorithm, so we've got to do it on the
  // host
  std::vector<gdf_size_type> indices(source_table->num_rows());
  std::iota(indices.begin(), indices.end(), 0);
  std::shuffle(indices.begin(), indices.end(), std::default_random_engine{});
  thrust::device_vector<gdf_size_type> target_indices(indices);

  std::shuffle(indices.begin(), indices.end(), std::default_random_engine{});
  thrust::device_vector<gdf_size_type> source_indices(indices);

  // Copy source_table row @ source_indices[i] to target_table @
  // target_indices[i]
  EXPECT_NO_THROW(thrust::for_each(
      rmm::exec_policy()->on(0),
      thrust::make_zip_iterator(
          thrust::make_tuple(target_indices.begin(), source_indices.begin())),
      thrust::make_zip_iterator(
          thrust::make_tuple(target_indices.end(), source_indices.end())),
      row_copier<true>{*target_table, *source_table}));

  // ensure source_table row @ source_indices[i] == target_table row @
  // target_indices[i] regardless of NULL ?= NULL
  EXPECT_TRUE(thrust::all_of(
      rmm::exec_policy()->on(0),
      thrust::make_zip_iterator(
          thrust::make_tuple(source_indices.begin(), target_indices.begin())),
      thrust::make_zip_iterator(
          thrust::make_tuple(source_indices.end(), target_indices.end())),
      row_comparison<true>{*source_table, *target_table, true}));

  EXPECT_TRUE(thrust::all_of(
      rmm::exec_policy()->on(0),
      thrust::make_zip_iterator(
          thrust::make_tuple(source_indices.begin(), target_indices.begin())),
      thrust::make_zip_iterator(
          thrust::make_tuple(source_indices.end(), target_indices.end())),
      row_comparison<true>{*source_table, *target_table, false}));

  // No row should contain a null value
  cudf::table target_host_table{target_cols.data(),
                                static_cast<gdf_size_type>(target_cols.size())};
  auto row_bitmask = cudf::row_bitmask(target_host_table);

  EXPECT_TRUE(thrust::all_of(
      rmm::exec_policy()->on(0), thrust::make_counting_iterator(0),
      thrust::make_counting_iterator(target_host_table.num_rows()),
      verify_bitmask{row_bitmask.data().get()}));
}

TEST_F(DeviceTableTest, CopyRowsSourceNoBitmaskTargetNull) {
  auto init_values = [](auto index) { return index; };

  cudf::test::column_wrapper<int32_t> source_col0(size, init_values);
  cudf::test::column_wrapper<float> source_col1(size, init_values);
  cudf::test::column_wrapper<double> source_col2(size, init_values);
  cudf::test::column_wrapper<int8_t> source_col3(size, init_values);
  std::vector<gdf_column*> source_cols{source_col0, source_col1, source_col2,
                                       source_col3};
  auto source_table =
      device_table::create(source_cols.size(), source_cols.data());

  cudf::test::column_wrapper<int32_t> target_col0(size, true);
  cudf::test::column_wrapper<float> target_col1(size, true);
  cudf::test::column_wrapper<double> target_col2(size, true);
  cudf::test::column_wrapper<int8_t> target_col3(size, true);
  std::vector<gdf_column*> target_cols{target_col0, target_col1, target_col2,
                                       target_col3};

  auto target_table =
      device_table::create(target_cols.size(), target_cols.data());

  // Copy a random row from the source table to a random row in the target table
  // Thrust doesn't have a `shuffle` algorithm, so we've got to do it on the
  // host
  std::vector<gdf_size_type> indices(source_table->num_rows());
  std::iota(indices.begin(), indices.end(), 0);
  std::shuffle(indices.begin(), indices.end(), std::default_random_engine{});
  thrust::device_vector<gdf_size_type> target_indices(indices);

  std::shuffle(indices.begin(), indices.end(), std::default_random_engine{});
  thrust::device_vector<gdf_size_type> source_indices(indices);

  // Copy source_table row @ source_indices[i] to target_table @
  // target_indices[i]
  EXPECT_NO_THROW(thrust::for_each(
      rmm::exec_policy()->on(0),
      thrust::make_zip_iterator(
          thrust::make_tuple(target_indices.begin(), source_indices.begin())),
      thrust::make_zip_iterator(
          thrust::make_tuple(target_indices.end(), source_indices.end())),
      row_copier<true>{*target_table, *source_table}));

  // ensure source_table row @ source_indices[i] == target_table row @
  // target_indices[i] regardless of NULL ?= NULL
  EXPECT_TRUE(thrust::all_of(
      rmm::exec_policy()->on(0),
      thrust::make_zip_iterator(
          thrust::make_tuple(source_indices.begin(), target_indices.begin())),
      thrust::make_zip_iterator(
          thrust::make_tuple(source_indices.end(), target_indices.end())),
      row_comparison<true>{*source_table, *target_table, true}));

  EXPECT_TRUE(thrust::all_of(
      rmm::exec_policy()->on(0),
      thrust::make_zip_iterator(
          thrust::make_tuple(source_indices.begin(), target_indices.begin())),
      thrust::make_zip_iterator(
          thrust::make_tuple(source_indices.end(), target_indices.end())),
      row_comparison<true>{*source_table, *target_table, false}));

  // No row should contain a null value
  cudf::table target_host_table{target_cols.data(),
                                static_cast<gdf_size_type>(target_cols.size())};
  auto row_bitmask = cudf::row_bitmask(target_host_table);

  EXPECT_TRUE(thrust::all_of(
      rmm::exec_policy()->on(0), thrust::make_counting_iterator(0),
      thrust::make_counting_iterator(target_host_table.num_rows()),
      verify_bitmask{row_bitmask.data().get()}));
}