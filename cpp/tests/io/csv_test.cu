#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <tests/utilities/base_fixture.hpp>
#include <tests/utilities/column_utilities.hpp>
#include <tests/utilities/column_wrapper.hpp>
#include <tests/utilities/cudf_gtest.hpp>
#include <tests/utilities/type_lists.hpp>

#include <cudf/io/functions.hpp>
#include <cudf/strings/string_view.cuh>
#include <cudf/strings/strings_column_view.hpp>
#include <cudf/table/table.hpp>
#include <cudf/table/table_view.hpp>

#include <arrow/io/api.h>
#include <gmock/gmock.h>

#include <algorithm>
#include <fstream>
#include <iostream>
#include <iterator>
#include <numeric>
#include <sstream>
#include <string>
#include <vector>

namespace cudf_io = cudf::experimental::io;

template <typename T>
using column_wrapper = typename std::conditional<std::is_same<T, cudf::string_view>::value,
                                                 cudf::test::strings_column_wrapper,
                                                 cudf::test::fixed_width_column_wrapper<T>>::type;
using column         = cudf::column;
using table          = cudf::experimental::table;
using table_view     = cudf::table_view;

// Global environment for temporary files
auto const temp_env = static_cast<cudf::test::TempDirTestEnvironment*>(
  ::testing::AddGlobalTestEnvironment(new cudf::test::TempDirTestEnvironment));

// Base test fixture for tests
struct CsvReaderTest : public cudf::test::BaseFixture {
};

// Typed test fixture for timestamp type tests
template <typename T>
struct CsvReaderNumericTypeTest : public CsvReaderTest {
  auto type() { return cudf::data_type{cudf::experimental::type_to_id<T>()}; }
};

// Declare typed test cases
using SupportedNumericTypes = cudf::test::Types<int64_t, double>;
TYPED_TEST_CASE(CsvReaderNumericTypeTest, SupportedNumericTypes);

namespace {
// Generates a vector of uniform random values of type T
template <typename T>
inline auto random_values(size_t size)
{
  std::vector<T> values(size);

  using T1 = T;
  using uniform_distribution =
    typename std::conditional_t<std::is_same<T1, bool>::value,
                                std::bernoulli_distribution,
                                std::conditional_t<std::is_floating_point<T1>::value,
                                                   std::uniform_real_distribution<T1>,
                                                   std::uniform_int_distribution<T1>>>;

  static constexpr auto seed = 0xf00d;
  static std::mt19937 engine{seed};
  static uniform_distribution dist{};
  std::generate_n(values.begin(), size, [&]() { return T{dist(engine)}; });

  return values;
}

MATCHER_P(FloatNearPointwise, tolerance, "Out-of-range")
{
  return (std::get<0>(arg) > std::get<1>(arg) - tolerance &&
          std::get<0>(arg) < std::get<1>(arg) + tolerance);
}

// Helper function to compare two floating-point column contents
template <typename T, typename std::enable_if_t<std::is_floating_point<T>::value>* = nullptr>
void expect_column_data_equal(std::vector<T> const& lhs, cudf::column_view const& rhs)
{
  EXPECT_THAT(cudf::test::to_host<T>(rhs).first,
              ::testing::Pointwise(FloatNearPointwise(1e-6), lhs));
}

// Helper function to compare two column contents
template <typename T, typename std::enable_if_t<!std::is_floating_point<T>::value>* = nullptr>
void expect_column_data_equal(std::vector<T> const& lhs, cudf::column_view const& rhs)
{
  EXPECT_THAT(cudf::test::to_host<T>(rhs).first, ::testing::ElementsAreArray(lhs));
}

}  // namespace

TYPED_TEST(CsvReaderNumericTypeTest, SingleColumn)
{
  constexpr auto num_rows = 10;
  auto sequence           = cudf::test::make_counting_transform_iterator(
    0, [](auto i) { return static_cast<TypeParam>(i + 1000.50f); });

  auto filepath = temp_env->get_temp_filepath("SingleColumn.csv");
  {
    std::ofstream out_file{filepath, std::ofstream::out};
    std::ostream_iterator<TypeParam> output_iterator(out_file, "\n");
    std::copy(sequence, sequence + num_rows, output_iterator);
  }

  cudf_io::read_csv_args in_args{cudf_io::source_info{filepath}};
  in_args.header = -1;
  auto result    = cudf_io::read_csv(in_args);

  const auto view = result.tbl->view();
  expect_column_data_equal(std::vector<TypeParam>(sequence, sequence + num_rows), view.column(0));
}

TEST_F(CsvReaderTest, MultiColumn)
{
  constexpr auto num_rows = 10;
  auto int8_values        = random_values<int8_t>(num_rows);
  auto int16_values       = random_values<int16_t>(num_rows);
  auto int32_values       = random_values<int32_t>(num_rows);
  auto int64_values       = random_values<int64_t>(num_rows);
  auto float32_values     = random_values<float>(num_rows);
  auto float64_values     = random_values<double>(num_rows);

  auto filepath = temp_env->get_temp_dir() + "MultiColumn.csv";
  {
    std::ostringstream line;
    for (int i = 0; i < num_rows; ++i) {
      line << std::to_string(int8_values[i]) << "," << int16_values[i] << "," << int16_values[i]
           << "," << int32_values[i] << "," << int32_values[i] << "," << int64_values[i] << ","
           << int64_values[i] << "," << float32_values[i] << "," << float32_values[i] << ","
           << float64_values[i] << "," << float64_values[i] << "\n";
    }
    std::ofstream outfile(filepath, std::ofstream::out);
    outfile << line.str();
  }

  cudf_io::read_csv_args in_args{cudf_io::source_info{filepath}};
  in_args.dtype  = {"int8",
                   "short",
                   "int16",
                   "int",
                   "int32",
                   "long",
                   "int64",
                   "float",
                   "float32",
                   "double",
                   "float64"};
  in_args.header = -1;
  auto result    = cudf_io::read_csv(in_args);

  const auto view = result.tbl->view();
  expect_column_data_equal(int8_values, view.column(0));
  expect_column_data_equal(int16_values, view.column(1));
  expect_column_data_equal(int16_values, view.column(2));
  expect_column_data_equal(int32_values, view.column(3));
  expect_column_data_equal(int32_values, view.column(4));
  expect_column_data_equal(int64_values, view.column(5));
  expect_column_data_equal(int64_values, view.column(6));
  expect_column_data_equal(float32_values, view.column(7));
  expect_column_data_equal(float32_values, view.column(8));
  expect_column_data_equal(float64_values, view.column(9));
  expect_column_data_equal(float64_values, view.column(10));
}

TEST_F(CsvReaderTest, Booleans)
{
  auto filepath = temp_env->get_temp_dir() + "Booleans.csv";
  {
    std::ofstream outfile(filepath, std::ofstream::out);
    outfile << "YES,1,bar,true\nno,2,FOO,true\nBar,3,yes,false\nNo,4,NO,"
               "true\nYes,5,foo,false\n";
  }

  cudf_io::read_csv_args in_args{cudf_io::source_info{filepath}};
  in_args.names        = {"A", "B", "C", "D"};
  in_args.dtype        = {"int32", "int32", "short", "bool"};
  in_args.true_values  = {"yes", "Yes", "YES", "foo", "FOO"};
  in_args.false_values = {"no", "No", "NO", "Bar", "bar"};
  in_args.header       = -1;
  auto result          = cudf_io::read_csv(in_args);

  // Booleans are the same (integer) data type, but valued at 0 or 1
  const auto view = result.tbl->view();
  EXPECT_EQ(4, view.num_columns());
  ASSERT_EQ(cudf::type_id::INT32, view.column(0).type().id());
  ASSERT_EQ(cudf::type_id::INT32, view.column(1).type().id());
  ASSERT_EQ(cudf::type_id::INT16, view.column(2).type().id());
  ASSERT_EQ(cudf::type_id::BOOL8, view.column(3).type().id());

  expect_column_data_equal(std::vector<int32_t>{1, 0, 0, 0, 1}, view.column(0));
  expect_column_data_equal(std::vector<int16_t>{0, 1, 1, 0, 1}, view.column(2));
  expect_column_data_equal(std::vector<bool>{true, true, false, true, false}, view.column(3));
}

TEST_F(CsvReaderTest, Dates)
{
  auto filepath = temp_env->get_temp_dir() + "Dates.csv";
  {
    std::ofstream outfile(filepath, std::ofstream::out);
    outfile << "05/03/2001\n31/10/2010\n20/10/1994\n18/10/1990\n1/1/1970\n";
    outfile << "18/04/1995\n14/07/1994\n07/06/2006 11:20:30.400\n";
    outfile << "16/09/2005T1:2:30.400PM\n2/2/1970\n";
  }

  cudf_io::read_csv_args in_args{cudf_io::source_info{filepath}};
  in_args.names    = {"A"};
  in_args.dtype    = {"date"};
  in_args.dayfirst = true;
  in_args.header   = -1;
  auto result      = cudf_io::read_csv(in_args);

  const auto view = result.tbl->view();
  EXPECT_EQ(1, view.num_columns());
  ASSERT_EQ(cudf::type_id::TIMESTAMP_MILLISECONDS, view.column(0).type().id());

  expect_column_data_equal(std::vector<cudf::timestamp_ms>{983750400000,
                                                           1288483200000,
                                                           782611200000,
                                                           656208000000,
                                                           0,
                                                           798163200000,
                                                           774144000000,
                                                           1149679230400,
                                                           1126875750400,
                                                           2764800000},
                           view.column(0));
}

TEST_F(CsvReaderTest, DatesCastToTimestampSeconds)
{
  auto filepath = temp_env->get_temp_dir() + "DatesCastToTimestampS.csv";
  {
    std::ofstream outfile(filepath, std::ofstream::out);
    outfile << "05/03/2001\n31/10/2010\n20/10/1994\n18/10/1990\n1/1/1970\n";
    outfile << "18/04/1995\n14/07/1994\n07/06/2006 11:20:30.400\n";
    outfile << "16/09/2005T1:2:30.400PM\n2/2/1970\n";
  }

  cudf_io::read_csv_args in_args{cudf_io::source_info{filepath}};
  in_args.names          = {"A"};
  in_args.dtype          = {"date"};
  in_args.dayfirst       = true;
  in_args.header         = -1;
  in_args.timestamp_type = cudf::data_type{cudf::type_id::TIMESTAMP_SECONDS};
  auto result            = cudf_io::read_csv(in_args);

  const auto view = result.tbl->view();
  EXPECT_EQ(1, view.num_columns());
  ASSERT_EQ(cudf::type_id::TIMESTAMP_SECONDS, view.column(0).type().id());

  expect_column_data_equal(std::vector<cudf::timestamp_s>{983750400,
                                                          1288483200,
                                                          782611200,
                                                          656208000,
                                                          0,
                                                          798163200,
                                                          774144000,
                                                          1149679230,
                                                          1126875750,
                                                          2764800},
                           view.column(0));
}

TEST_F(CsvReaderTest, DatesCastToTimestampMilliSeconds)
{
  auto filepath = temp_env->get_temp_dir() + "DatesCastToTimestampMs.csv";
  {
    std::ofstream outfile(filepath, std::ofstream::out);
    outfile << "05/03/2001\n31/10/2010\n20/10/1994\n18/10/1990\n1/1/1970\n";
    outfile << "18/04/1995\n14/07/1994\n07/06/2006 11:20:30.400\n";
    outfile << "16/09/2005T1:2:30.400PM\n2/2/1970\n";
  }

  cudf_io::read_csv_args in_args{cudf_io::source_info{filepath}};
  in_args.names          = {"A"};
  in_args.dtype          = {"date"};
  in_args.dayfirst       = true;
  in_args.header         = -1;
  in_args.timestamp_type = cudf::data_type{cudf::type_id::TIMESTAMP_MILLISECONDS};
  auto result            = cudf_io::read_csv(in_args);

  const auto view = result.tbl->view();
  EXPECT_EQ(1, view.num_columns());
  ASSERT_EQ(cudf::type_id::TIMESTAMP_MILLISECONDS, view.column(0).type().id());

  expect_column_data_equal(std::vector<cudf::timestamp_ms>{983750400000,
                                                           1288483200000,
                                                           782611200000,
                                                           656208000000,
                                                           0,
                                                           798163200000,
                                                           774144000000,
                                                           1149679230400,
                                                           1126875750400,
                                                           2764800000},
                           view.column(0));
}

TEST_F(CsvReaderTest, DatesCastToTimestampMicroSeconds)
{
  auto filepath = temp_env->get_temp_dir() + "DatesCastToTimestampUs.csv";
  {
    std::ofstream outfile(filepath, std::ofstream::out);
    outfile << "05/03/2001\n31/10/2010\n20/10/1994\n18/10/1990\n1/1/1970\n";
    outfile << "18/04/1995\n14/07/1994\n07/06/2006 11:20:30.400\n";
    outfile << "16/09/2005T1:2:30.400PM\n2/2/1970\n";
  }

  cudf_io::read_csv_args in_args{cudf_io::source_info{filepath}};
  in_args.names          = {"A"};
  in_args.dtype          = {"date"};
  in_args.dayfirst       = true;
  in_args.header         = -1;
  in_args.timestamp_type = cudf::data_type{cudf::type_id::TIMESTAMP_MICROSECONDS};
  auto result            = cudf_io::read_csv(in_args);

  const auto view = result.tbl->view();
  EXPECT_EQ(1, view.num_columns());
  ASSERT_EQ(cudf::type_id::TIMESTAMP_MICROSECONDS, view.column(0).type().id());

  expect_column_data_equal(std::vector<cudf::timestamp_us>{983750400000000,
                                                           1288483200000000,
                                                           782611200000000,
                                                           656208000000000,
                                                           0,
                                                           798163200000000,
                                                           774144000000000,
                                                           1149679230400000,
                                                           1126875750400000,
                                                           2764800000000},
                           view.column(0));
}

TEST_F(CsvReaderTest, DatesCastToTimestampNanoSeconds)
{
  auto filepath = temp_env->get_temp_dir() + "DatesCastToTimestampNs.csv";
  {
    std::ofstream outfile(filepath, std::ofstream::out);
    outfile << "05/03/2001\n31/10/2010\n20/10/1994\n18/10/1990\n1/1/1970\n";
    outfile << "18/04/1995\n14/07/1994\n07/06/2006 11:20:30.400\n";
    outfile << "16/09/2005T1:2:30.400PM\n2/2/1970\n";
  }

  cudf_io::read_csv_args in_args{cudf_io::source_info{filepath}};
  in_args.names          = {"A"};
  in_args.dtype          = {"date"};
  in_args.dayfirst       = true;
  in_args.header         = -1;
  in_args.timestamp_type = cudf::data_type{cudf::type_id::TIMESTAMP_NANOSECONDS};
  auto result            = cudf_io::read_csv(in_args);

  const auto view = result.tbl->view();
  EXPECT_EQ(1, view.num_columns());
  ASSERT_EQ(cudf::type_id::TIMESTAMP_NANOSECONDS, view.column(0).type().id());

  expect_column_data_equal(std::vector<cudf::timestamp_ns>{983750400000000000,
                                                           1288483200000000000,
                                                           782611200000000000,
                                                           656208000000000000,
                                                           0,
                                                           798163200000000000,
                                                           774144000000000000,
                                                           1149679230400000000,
                                                           1126875750400000000,
                                                           2764800000000000},
                           view.column(0));
}

TEST_F(CsvReaderTest, FloatingPoint)
{
  auto filepath = temp_env->get_temp_dir() + "FloatingPoint.csv";
  {
    std::ofstream outfile(filepath, std::ofstream::out);
    outfile << "5.6;0.5679e2;1.2e10;0.07e1;3000e-3;12.34e0;3.1e-001;-73."
               "98007199999998;";
  }

  cudf_io::read_csv_args in_args{cudf_io::source_info{filepath}};
  in_args.names          = {"A"};
  in_args.dtype          = {"float32"};
  in_args.lineterminator = ';';
  in_args.header         = -1;
  auto result            = cudf_io::read_csv(in_args);

  const auto view = result.tbl->view();
  EXPECT_EQ(1, view.num_columns());
  ASSERT_EQ(cudf::type_id::FLOAT32, view.column(0).type().id());

  const auto ref_vals =
    std::vector<float>{5.6, 56.79, 12000000000, 0.7, 3.000, 12.34, 0.31, -73.98007199999998};
  expect_column_data_equal(ref_vals, view.column(0));

  const auto bitmask = cudf::test::bitmask_to_host(view.column(0));
  ASSERT_EQ((1u << ref_vals.size()) - 1, bitmask[0]);
}

TEST_F(CsvReaderTest, Strings)
{
  std::vector<std::string> names{"line", "verse"};

  auto filepath = temp_env->get_temp_dir() + "Strings.csv";
  {
    std::ofstream outfile(filepath, std::ofstream::out);
    outfile << names[0] << ',' << names[1] << ',' << '\n';
    outfile << "10,abc def ghi" << '\n';
    outfile << "20,\"jkl mno pqr\"" << '\n';
    outfile << "30,stu \"\"vwx\"\" yz" << '\n';
  }

  cudf_io::read_csv_args in_args{cudf_io::source_info{filepath}};
  in_args.names   = names;
  in_args.dtype   = {"int32", "str"};
  in_args.quoting = cudf_io::quote_style::NONE;
  auto result     = cudf_io::read_csv(in_args);

  const auto view = result.tbl->view();
  EXPECT_EQ(2, view.num_columns());
  ASSERT_EQ(cudf::type_id::INT32, view.column(0).type().id());
  ASSERT_EQ(cudf::type_id::STRING, view.column(1).type().id());

  expect_column_data_equal(
    std::vector<std::string>{"abc def ghi", "\"jkl mno pqr\"", "stu \"\"vwx\"\" yz"},
    view.column(1));
}

TEST_F(CsvReaderTest, DISABLED_StringsQuotes)
{
  std::vector<std::string> names{"line", "verse"};

  auto filepath = temp_env->get_temp_dir() + "StringsQuotes.csv";
  {
    std::ofstream outfile(filepath, std::ofstream::out);
    outfile << names[0] << ',' << names[1] << ',' << '\n';
    outfile << "10,`abc,\ndef, ghi`" << '\n';
    outfile << "20,`jkl, ``mno``, pqr`" << '\n';
    outfile << "30,stu `vwx` yz" << '\n';
  }

  cudf_io::read_csv_args in_args{cudf_io::source_info{filepath}};
  in_args.names     = names;
  in_args.dtype     = {"int32", "str"};
  in_args.quotechar = '`';
  auto result       = cudf_io::read_csv(in_args);

  const auto view = result.tbl->view();
  EXPECT_EQ(2, view.num_columns());
  ASSERT_EQ(cudf::type_id::INT32, view.column(0).type().id());
  ASSERT_EQ(cudf::type_id::STRING, view.column(1).type().id());

  expect_column_data_equal(
    std::vector<std::string>{"abc,\ndef, ghi", "jkl, `mno`, pqr", "stu `vwx` yz"}, view.column(1));
}

TEST_F(CsvReaderTest, StringsQuotesIgnored)
{
  std::vector<std::string> names{"line", "verse"};

  auto filepath = temp_env->get_temp_dir() + "StringsQuotesIgnored.csv";
  {
    std::ofstream outfile(filepath, std::ofstream::out);
    outfile << names[0] << ',' << names[1] << ',' << '\n';
    outfile << "10,\"abcdef ghi\"" << '\n';
    outfile << "20,\"jkl \"\"mno\"\" pqr\"" << '\n';
    outfile << "30,stu \"vwx\" yz" << '\n';
  }

  cudf_io::read_csv_args in_args{cudf_io::source_info{filepath}};
  in_args.names       = names;
  in_args.dtype       = {"int32", "str"};
  in_args.quoting     = cudf_io::quote_style::NONE;
  in_args.doublequote = false;  // do not replace double quotechar with single
  auto result         = cudf_io::read_csv(in_args);

  const auto view = result.tbl->view();
  EXPECT_EQ(2, view.num_columns());
  ASSERT_EQ(cudf::type_id::INT32, view.column(0).type().id());
  ASSERT_EQ(cudf::type_id::STRING, view.column(1).type().id());

  expect_column_data_equal(
    std::vector<std::string>{"\"abcdef ghi\"", "\"jkl \"\"mno\"\" pqr\"", "stu \"vwx\" yz"},
    view.column(1));
}

TEST_F(CsvReaderTest, SkiprowsNrows)
{
  auto filepath = temp_env->get_temp_dir() + "SkiprowsNrows.csv";
  {
    std::ofstream outfile(filepath, std::ofstream::out);
    outfile << "1\n2\n3\n4\n5\n6\n7\n8\n9\n";
  }

  cudf_io::read_csv_args in_args{cudf_io::source_info{filepath}};
  in_args.names      = {"A"};
  in_args.dtype      = {"int32"};
  in_args.header     = 1;
  in_args.skiprows   = 2;
  in_args.skipfooter = 0;
  in_args.nrows      = 2;
  auto result        = cudf_io::read_csv(in_args);

  const auto view = result.tbl->view();
  EXPECT_EQ(1, view.num_columns());
  ASSERT_EQ(cudf::type_id::INT32, view.column(0).type().id());

  expect_column_data_equal(std::vector<int32_t>{5, 6}, view.column(0));
}

TEST_F(CsvReaderTest, ByteRange)
{
  auto filepath = temp_env->get_temp_dir() + "ByteRange.csv";
  {
    std::ofstream outfile(filepath, std::ofstream::out);
    outfile << "1000\n2000\n3000\n4000\n5000\n6000\n7000\n8000\n9000\n";
  }

  cudf_io::read_csv_args in_args{cudf_io::source_info{filepath}};
  in_args.names             = {"A"};
  in_args.dtype             = {"int32"};
  in_args.header            = -1;
  in_args.byte_range_offset = 11;
  in_args.byte_range_size   = 15;
  auto result               = cudf_io::read_csv(in_args);

  const auto view = result.tbl->view();
  EXPECT_EQ(1, view.num_columns());
  ASSERT_EQ(cudf::type_id::INT32, view.column(0).type().id());

  expect_column_data_equal(std::vector<int32_t>{4000, 5000, 6000}, view.column(0));
}

TEST_F(CsvReaderTest, ByteRangeStrings)
{
  std::string input = "\"a\"\n\"b\"\n\"c\"";
  cudf_io::read_csv_args in_args{cudf_io::source_info{input.c_str(), input.size()}};
  in_args.names             = {"A"};
  in_args.dtype             = {"str"};
  in_args.header            = -1;
  in_args.byte_range_offset = 4;
  auto result               = cudf_io::read_csv(in_args);

  const auto view = result.tbl->view();
  EXPECT_EQ(1, view.num_columns());
  ASSERT_EQ(cudf::type_id::STRING, view.column(0).type().id());

  expect_column_data_equal(std::vector<std::string>{"c"}, view.column(0));
}

TEST_F(CsvReaderTest, BlanksAndComments)
{
  auto filepath = temp_env->get_temp_dir() + "BlanksAndComments.csv";
  {
    std::ofstream outfile(filepath, std::ofstream::out);
    outfile << "1\n#blank\n3\n4\n5\n#blank\n\n\n8\n9\n";
  }

  cudf_io::read_csv_args in_args{cudf_io::source_info{filepath}};
  in_args.names   = {"A"};
  in_args.dtype   = {"int32"};
  in_args.header  = -1;
  in_args.comment = '#';
  auto result     = cudf_io::read_csv(in_args);

  const auto view = result.tbl->view();
  EXPECT_EQ(1, view.num_columns());
  ASSERT_EQ(cudf::type_id::INT32, view.column(0).type().id());

  expect_column_data_equal(std::vector<int32_t>{1, 3, 4, 5, 8, 9}, view.column(0));
}

TEST_F(CsvReaderTest, EmptyFile)
{
  auto filepath = temp_env->get_temp_dir() + "EmptyFile.csv";
  {
    std::ofstream outfile{filepath, std::ofstream::out};
    outfile << "";
  }

  cudf_io::read_csv_args in_args{cudf_io::source_info{filepath}};
  auto result = cudf_io::read_csv(in_args);

  const auto view = result.tbl->view();
  EXPECT_EQ(0, view.num_columns());
}

TEST_F(CsvReaderTest, NoDataFile)
{
  auto filepath = temp_env->get_temp_dir() + "NoDataFile.csv";
  {
    std::ofstream outfile{filepath, std::ofstream::out};
    outfile << "\n\n";
  }

  cudf_io::read_csv_args in_args{cudf_io::source_info{filepath}};
  auto result = cudf_io::read_csv(in_args);

  const auto view = result.tbl->view();
  EXPECT_EQ(0, view.num_columns());
}

TEST_F(CsvReaderTest, ArrowFileSource)
{
  auto filepath = temp_env->get_temp_dir() + "ArrowFileSource.csv";
  {
    std::ofstream outfile(filepath, std::ofstream::out);
    outfile << "A\n9\n8\n7\n6\n5\n4\n3\n2\n";
  }

  std::shared_ptr<arrow::io::ReadableFile> infile;
  ASSERT_TRUE(arrow::io::ReadableFile::Open(filepath, &infile).ok());

  cudf_io::read_csv_args in_args{cudf_io::source_info{infile}};
  in_args.dtype = {"int8"};
  auto result   = cudf_io::read_csv(in_args);

  const auto view = result.tbl->view();
  EXPECT_EQ(1, view.num_columns());
  ASSERT_EQ(cudf::type_id::INT8, view.column(0).type().id());

  expect_column_data_equal(std::vector<int8_t>{9, 8, 7, 6, 5, 4, 3, 2}, view.column(0));
}

TEST_F(CsvReaderTest, InvalidFloatingPoint)
{
  const auto filepath = temp_env->get_temp_dir() + "InvalidFloatingPoint.csv";
  {
    std::ofstream outfile(filepath, std::ofstream::out);
    outfile << "1.2e1+\n3.4e2-\n5.6e3e\n7.8e3A\n9.0Be1\n1C.2";
  }

  cudf_io::read_csv_args in_args{cudf_io::source_info{filepath}};
  in_args.names     = {"A"};
  in_args.dtype     = {"float32"};
  in_args.header    = -1;
  const auto result = cudf_io::read_csv(in_args);

  const auto view = result.tbl->view();
  EXPECT_EQ(1, view.num_columns());
  ASSERT_EQ(cudf::type_id::FLOAT32, view.column(0).type().id());

  const auto col_data = cudf::test::to_host<float>(view.column(0));
  // col_data.first contains the column data
  for (const auto& elem : col_data.first) ASSERT_TRUE(std::isnan(elem));
  // col_data.second contains the bitmasks
  ASSERT_EQ(0u, col_data.second[0]);
}

TEST_F(CsvReaderTest, StringInference)
{
  std::string buffer = "\"-1\"\n";
  cudf_io::read_csv_args in_args{cudf_io::source_info{buffer.c_str(), buffer.size()}};
  in_args.header    = -1;
  const auto result = cudf_io::read_csv(in_args);

  EXPECT_EQ(result.tbl->num_columns(), 1);
  EXPECT_EQ(result.tbl->get_column(0).type().id(), cudf::STRING);
}

CUDF_TEST_PROGRAM_MAIN()
