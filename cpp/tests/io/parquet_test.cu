/*
 * Copyright (c) 2019, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <tests/utilities/base_fixture.hpp>
#include <tests/utilities/column_utilities.hpp>
#include <tests/utilities/column_wrapper.hpp>
#include <tests/utilities/cudf_gtest.hpp>
#include <tests/utilities/type_lists.hpp>

#include <cudf/io/functions.hpp>
#include <cudf/strings/string_view.cuh>
#include <cudf/strings/strings_column_view.hpp>
#include <cudf/table/table.hpp>
#include <cudf/table/table_view.hpp>

#include <type_traits>

namespace cudf_io = cudf::experimental::io;

template <typename T>
using column_wrapper =
    typename std::conditional<std::is_same<T, cudf::string_view>::value,
                              cudf::test::strings_column_wrapper,
                              cudf::test::fixed_width_column_wrapper<T>>::type;
using column = cudf::column;
using table = cudf::experimental::table;
using table_view = cudf::table_view;

// Global environment for temporary files
auto const temp_env = static_cast<cudf::test::TempDirTestEnvironment*>(
    ::testing::AddGlobalTestEnvironment(
        new cudf::test::TempDirTestEnvironment));

// Base test fixture for tests
struct ParquetWriterTest : public cudf::test::BaseFixture {};

// Typed test fixture for numeric type tests
template <typename T>
struct ParquetWriterNumericTypeTest : public ParquetWriterTest {
  auto type() {
    return cudf::data_type{cudf::experimental::type_to_id<T>()};
  }
};

// Typed test fixture for timestamp type tests
template <typename T>
struct ParquetWriterTimestampTypeTest : public ParquetWriterTest {
  auto type() {
    return cudf::data_type{cudf::experimental::type_to_id<T>()};
  }
};

// Declare typed test cases
TYPED_TEST_CASE(ParquetWriterNumericTypeTest, cudf::test::NumericTypes);
using SupportedTimestampTypes = cudf::test::TimestampTypes;
TYPED_TEST_CASE(ParquetWriterTimestampTypeTest, SupportedTimestampTypes);

namespace {

// Generates a vector of uniform random values of type T
template <typename T>
inline auto random_values(size_t size) {
  std::vector<T> values(size);

  using T1 = T;
  using uniform_distribution = typename std::conditional_t<
      std::is_same<T1, bool>::value, std::bernoulli_distribution,
      std::conditional_t<std::is_floating_point<T1>::value,
                         std::uniform_real_distribution<T1>,
                         std::uniform_int_distribution<T1>>>;

  static constexpr auto seed = 0xf00d;
  static std::mt19937 engine{seed};
  static uniform_distribution dist{};
  std::generate_n(values.begin(), size, [&]() { return T{dist(engine)}; });

  return values;
}

// Helper function to compare two tables
void expect_tables_equal(cudf::table_view const& lhs,
                         cudf::table_view const& rhs) {
  EXPECT_EQ(lhs.num_columns(), rhs.num_columns());
  auto expected = lhs.begin();
  auto result = rhs.begin();
  while (result != rhs.end()) {
    cudf::test::expect_columns_equal(*expected++, *result++);
  }
}

}  // namespace

TYPED_TEST(ParquetWriterNumericTypeTest, SingleColumn) {
  auto sequence = cudf::test::make_counting_transform_iterator(
      0, [](auto i) { return TypeParam(i); });
  auto validity = cudf::test::make_counting_transform_iterator(
      0, [](auto i) { return true; });

  constexpr auto num_rows = 100;
  column_wrapper<TypeParam> col(sequence, sequence + num_rows, validity);

  std::vector<std::unique_ptr<column>> cols;
  cols.push_back(col.release());
  auto expected = std::make_unique<table>(std::move(cols));
  EXPECT_EQ(1, expected->num_columns());

  auto filepath = temp_env->get_temp_filepath("SingleColumn.parquet");
  cudf_io::write_parquet_args out_args{cudf_io::sink_info{filepath},
                                   expected->view()};
  cudf_io::write_parquet(out_args);

  cudf_io::read_parquet_args in_args{cudf_io::source_info{filepath}};
  auto result = cudf_io::read_parquet(in_args);

  expect_tables_equal(expected->view(), result.tbl->view());
}

TYPED_TEST(ParquetWriterNumericTypeTest, SingleColumnWithNulls) {
  auto sequence = cudf::test::make_counting_transform_iterator(
      0, [](auto i) { return TypeParam(i); });
  auto validity = cudf::test::make_counting_transform_iterator(
      0, [](auto i) { return (i % 2); });

  constexpr auto num_rows = 100;
  column_wrapper<TypeParam> col(sequence, sequence + num_rows, validity);

  std::vector<std::unique_ptr<column>> cols;
  cols.push_back(col.release());
  auto expected = std::make_unique<table>(std::move(cols));
  EXPECT_EQ(1, expected->num_columns());

  auto filepath = temp_env->get_temp_filepath("SingleColumnWithNulls.parquet");
  cudf_io::write_parquet_args out_args{cudf_io::sink_info{filepath},
                                       expected->view()};
  cudf_io::write_parquet(out_args);

  cudf_io::read_parquet_args in_args{cudf_io::source_info{filepath}};
  auto result = cudf_io::read_parquet(in_args);

  expect_tables_equal(expected->view(), result.tbl->view());
}

TYPED_TEST(ParquetWriterTimestampTypeTest, Timestamps) {
  auto sequence = cudf::test::make_counting_transform_iterator(
      0, [](auto i) { return TypeParam((std::rand() / 10000) * 1000); });
  auto validity = cudf::test::make_counting_transform_iterator(
      0, [](auto i) { return true; });

  constexpr auto num_rows = 100;
  column_wrapper<TypeParam> col(sequence, sequence + num_rows, validity);

  std::vector<std::unique_ptr<column>> cols;
  cols.push_back(col.release());
  auto expected = std::make_unique<table>(std::move(cols));
  EXPECT_EQ(1, expected->num_columns());

  auto filepath = temp_env->get_temp_filepath("Timestamps.parquet");
  cudf_io::write_parquet_args out_args{cudf_io::sink_info{filepath},
                                       expected->view()};
  cudf_io::write_parquet(out_args);

  cudf_io::read_parquet_args in_args{cudf_io::source_info{filepath}};
  in_args.timestamp_type = this->type();
  auto result = cudf_io::read_parquet(in_args);

  expect_tables_equal(expected->view(), result.tbl->view());
}

TYPED_TEST(ParquetWriterTimestampTypeTest, TimestampsWithNulls) {
  auto sequence = cudf::test::make_counting_transform_iterator(
      0, [](auto i) { return TypeParam((std::rand() / 10000) * 1000); });
  auto validity = cudf::test::make_counting_transform_iterator(
      0, [](auto i) { return (i > 30) && (i < 60); });

  constexpr auto num_rows = 100;
  column_wrapper<TypeParam> col(sequence, sequence + num_rows, validity);

  std::vector<std::unique_ptr<column>> cols;
  cols.push_back(col.release());
  auto expected = std::make_unique<table>(std::move(cols));
  EXPECT_EQ(1, expected->num_columns());

  auto filepath = temp_env->get_temp_filepath("TimestampsWithNulls.parquet");
  cudf_io::write_parquet_args out_args{cudf_io::sink_info{filepath},
                                   expected->view()};
  cudf_io::write_parquet(out_args);

  cudf_io::read_parquet_args in_args{cudf_io::source_info{filepath}};
  in_args.timestamp_type = this->type();
  auto result = cudf_io::read_parquet(in_args);

  expect_tables_equal(expected->view(), result.tbl->view());
}

TEST_F(ParquetWriterTest, MultiColumn) {
  constexpr auto num_rows = 100;

  // auto col0_data = random_values<bool>(num_rows);
  auto col1_data = random_values<int8_t>(num_rows);
  auto col2_data = random_values<int16_t>(num_rows);
  auto col3_data = random_values<int32_t>(num_rows);
  auto col4_data = random_values<float>(num_rows);
  auto col5_data = random_values<double>(num_rows);
  auto validity = cudf::test::make_counting_transform_iterator(
      0, [](auto i) { return true; });

  // column_wrapper<bool> col0{
  //    col0_data.begin(), col0_data.end(), validity};
  column_wrapper<int8_t> col1{col1_data.begin(), col1_data.end(), validity};
  column_wrapper<int16_t> col2{col2_data.begin(), col2_data.end(), validity};
  column_wrapper<int32_t> col3{col3_data.begin(), col3_data.end(), validity};
  column_wrapper<float> col4{col4_data.begin(), col4_data.end(), validity};
  column_wrapper<double> col5{col5_data.begin(), col5_data.end(), validity};

  cudf_io::table_metadata expected_metadata;
  //expected_metadata.column_names.emplace_back("bools");
  expected_metadata.column_names.emplace_back("int8s");
  expected_metadata.column_names.emplace_back("int16s");
  expected_metadata.column_names.emplace_back("int32s");
  expected_metadata.column_names.emplace_back("floats");
  expected_metadata.column_names.emplace_back("doubles");

  std::vector<std::unique_ptr<column>> cols;
  // cols.push_back(col0.release());
  cols.push_back(col1.release());
  cols.push_back(col2.release());
  cols.push_back(col3.release());
  cols.push_back(col4.release());
  cols.push_back(col5.release());
  auto expected = std::make_unique<table>(std::move(cols));
  EXPECT_EQ(5, expected->num_columns());

  auto filepath = temp_env->get_temp_filepath("MultiColumn.parquet");
  cudf_io::write_parquet_args out_args{cudf_io::sink_info{filepath},
                                       expected->view(), &expected_metadata};
  cudf_io::write_parquet(out_args);

  cudf_io::read_parquet_args in_args{cudf_io::source_info{filepath}};
  auto result = cudf_io::read_parquet(in_args);

  expect_tables_equal(expected->view(), result.tbl->view());
  EXPECT_EQ(expected_metadata.column_names, result.metadata.column_names);
}

TEST_F(ParquetWriterTest, MultiColumnWithNulls) {
  constexpr auto num_rows = 100;

  // auto col0_data = random_values<bool>(num_rows);
  auto col1_data = random_values<int8_t>(num_rows);
  auto col2_data = random_values<int16_t>(num_rows);
  auto col3_data = random_values<int32_t>(num_rows);
  auto col4_data = random_values<float>(num_rows);
  auto col5_data = random_values<double>(num_rows);
  // auto col0_mask = cudf::test::make_counting_transform_iterator(
  //    0, [](auto i) { return (i % 2); });
  auto col1_mask = cudf::test::make_counting_transform_iterator(
      0, [](auto i) { return (i < 10); });
  auto col2_mask = cudf::test::make_counting_transform_iterator(
      0, [](auto i) { return true; });
  auto col3_mask = cudf::test::make_counting_transform_iterator(
      0, [](auto i) { return (i == (num_rows - 1)); });
  auto col4_mask = cudf::test::make_counting_transform_iterator(
      0, [](auto i) { return (i >= 40 || i <= 60); });
  auto col5_mask = cudf::test::make_counting_transform_iterator(
      0, [](auto i) { return (i > 80); });

  // column_wrapper<bool> col0{
  //    col0_data.begin(), col0_data.end(), col0_mask};
  column_wrapper<int8_t> col1{col1_data.begin(), col1_data.end(), col1_mask};
  column_wrapper<int16_t> col2{col2_data.begin(), col2_data.end(), col2_mask};
  column_wrapper<int32_t> col3{col3_data.begin(), col3_data.end(), col3_mask};
  column_wrapper<float> col4{col4_data.begin(), col4_data.end(), col4_mask};
  column_wrapper<double> col5{col5_data.begin(), col5_data.end(), col5_mask};

  cudf_io::table_metadata expected_metadata;
  //expected_metadata.column_names.emplace_back("bools");
  expected_metadata.column_names.emplace_back("int8s");
  expected_metadata.column_names.emplace_back("int16s");
  expected_metadata.column_names.emplace_back("int32s");
  expected_metadata.column_names.emplace_back("floats");
  expected_metadata.column_names.emplace_back("doubles");

  std::vector<std::unique_ptr<column>> cols;
  // cols.push_back(col0.release());
  cols.push_back(col1.release());
  cols.push_back(col2.release());
  cols.push_back(col3.release());
  cols.push_back(col4.release());
  cols.push_back(col5.release());
  auto expected = std::make_unique<table>(std::move(cols));
  EXPECT_EQ(5, expected->num_columns());

  auto filepath = temp_env->get_temp_filepath("MultiColumnWithNulls.parquet");
  cudf_io::write_parquet_args out_args{cudf_io::sink_info{filepath},
                                       expected->view(), &expected_metadata};
  cudf_io::write_parquet(out_args);

  cudf_io::read_parquet_args in_args{cudf_io::source_info{filepath}};
  auto result = cudf_io::read_parquet(in_args);

  expect_tables_equal(expected->view(), result.tbl->view());
  EXPECT_EQ(expected_metadata.column_names, result.metadata.column_names);
}

TEST_F(ParquetWriterTest, Strings) {
  std::vector<const char*> strings{"Monday", "Monday", "Friday", "Monday",
                                   "Friday", "Friday", "Friday", "Funday"};
  const auto num_rows = strings.size();

  auto seq_col0 = random_values<int>(num_rows);
  auto seq_col2 = random_values<float>(num_rows);
  auto validity = cudf::test::make_counting_transform_iterator(
      0, [](auto i) { return true; });

  column_wrapper<int> col0{seq_col0.begin(), seq_col0.end(), validity};
  column_wrapper<cudf::string_view> col1{strings.begin(), strings.end()};
  column_wrapper<float> col2{seq_col2.begin(), seq_col2.end(), validity};

  cudf_io::table_metadata expected_metadata;
  expected_metadata.column_names.emplace_back("col_other");
  expected_metadata.column_names.emplace_back("col_string");
  expected_metadata.column_names.emplace_back("col_another");

  std::vector<std::unique_ptr<column>> cols;
  cols.push_back(col0.release());
  cols.push_back(col1.release());
  cols.push_back(col2.release());
  auto expected = std::make_unique<table>(std::move(cols));
  EXPECT_EQ(3, expected->num_columns());

  auto filepath = temp_env->get_temp_filepath("Strings.parquet");
  cudf_io::write_parquet_args out_args{cudf_io::sink_info{filepath},
                                       expected->view(), &expected_metadata};
  cudf_io::write_parquet(out_args);

  cudf_io::read_parquet_args in_args{cudf_io::source_info{filepath}};
  auto result = cudf_io::read_parquet(in_args);

  expect_tables_equal(expected->view(), result.tbl->view());
  EXPECT_EQ(expected_metadata.column_names, result.metadata.column_names);
}
