/*
 * Copyright (c) 2018, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <cstdlib>
#include <iostream>
#include <fstream>
#include <vector>
#include <sys/stat.h>

#include "gtest/gtest.h"
#include "gmock/gmock.h"

#include <cudf.h>
#include <NVStrings.h>

MATCHER_P(FloatNearPointwise, tolerance, "Out of range")
{
    return (std::get<0>(arg)>std::get<1>(arg)-tolerance &&
            std::get<0>(arg)<std::get<1>(arg)+tolerance) ;
}

bool checkFile(const char *fname)
{
	struct stat st;
	return (stat(fname, &st) ? 0 : 1);
}

// DESCRIPTION: Simple test internal helper class to transfer cudf column data
// from device to host for test comparisons and debugging/development
template <typename T>
class gdf_host_column
{
public:
	gdf_host_column() = delete;
	explicit gdf_host_column(gdf_column* const col)
	{
		m_hostdata = std::vector<T>(col->size);
		hipMemcpy(m_hostdata.data(), col->data, sizeof(T) * col->size, hipMemcpyDeviceToHost);
	}

	auto hostdata() const -> const auto&
	{
		return m_hostdata;
	}
	void print() const
	{
		for (size_t i = 0; i < m_hostdata.size(); ++i)
		{
			std::cout.precision(17);
			std::cout << "[" << i << "]: value=" << m_hostdata[i] << "\n";
		}
	}

private:
	std::vector<T> m_hostdata;
};

TEST(gdf_csv_test, Numbers)
{
	const char* fname	= "/tmp/CsvNumbersTest.csv";
	const char* names[]	= { "A", "B", "C", "D", "E" };
	const char* types[]	= { "short", "int", "long", "float64", "float32" };

	std::ofstream outfile(fname, std::ofstream::out);
	outfile <<	" 10, 20, 30, 0.40, 50000\n"\
				"-11,-21,-31,-0.41,-51111\n"\
				" 12, 22, 32, 0.42, 52222\n"\
				"-13,-23,-33,-0.43,-53333\n";
	outfile.close();
	ASSERT_TRUE( checkFile(fname) );

	{
		csv_read_arg args{};
		args.input_data_form    = gdf_csv_input_form::FILE_PATH;
		args.filepath_or_buffer = fname;
		args.num_cols           = std::extent<decltype(names)>::value;
		args.names              = names;
		args.dtype              = types;
		args.delimiter          = ',';
		args.lineterminator     = '\n';
		args.decimal            = '.';
		args.nrows              = -1;
		EXPECT_EQ( read_csv(&args), GDF_SUCCESS );

		ASSERT_EQ( args.data[0]->dtype, GDF_INT16 );
		ASSERT_EQ( args.data[1]->dtype, GDF_INT32 );
		ASSERT_EQ( args.data[2]->dtype, GDF_INT64 );
		ASSERT_EQ( args.data[3]->dtype, GDF_FLOAT64 );
		ASSERT_EQ( args.data[4]->dtype, GDF_FLOAT32 );
		auto ACol = gdf_host_column<int16_t>(args.data[0]);
		auto BCol = gdf_host_column<int32_t>(args.data[1]);
		auto CCol = gdf_host_column<int64_t>(args.data[2]);
		auto DCol = gdf_host_column<double>(args.data[3]);
		auto ECol = gdf_host_column<float>(args.data[4]);
		EXPECT_THAT( ACol.hostdata(), ::testing::ElementsAre<int16_t>(10, -11, 12, -13) );
		EXPECT_THAT( BCol.hostdata(), ::testing::ElementsAre<int32_t>(20, -21, 22, -23) );
		EXPECT_THAT( CCol.hostdata(), ::testing::ElementsAre<int64_t>(30, -31, 32, -33) );
		EXPECT_THAT( DCol.hostdata(), ::testing::ElementsAre<double>(0.40, -0.41, 0.42, -0.43) );
		EXPECT_THAT( ECol.hostdata(), ::testing::ElementsAre<float>(50000, -51111, 52222, -53333) );
	}
}

TEST(gdf_csv_test, MortPerf)
{
	gdf_error error = GDF_SUCCESS;

	csv_read_arg	args{};
	const int num_cols = 31;

    args.num_cols = num_cols;
	args.nrows = -1;

    const char ** dnames = new const char *[num_cols] {
        "loan_id",
        "monthly_reporting_period",
        "servicer",
        "interest_rate",
        "current_actual_upb",
        "loan_age",
        "remaining_months_to_legal_maturity",
        "adj_remaining_months_to_maturity",
        "maturity_date",
        "msa",
        "current_loan_delinquency_status",
        "mod_flag",
        "zero_balance_code",
        "zero_balance_effective_date",
        "last_paid_installment_date",
        "foreclosed_after",
        "disposition_date",
        "foreclosure_costs",
        "prop_preservation_and_repair_costs",
        "asset_recovery_costs",
        "misc_holding_expenses",
        "holding_taxes",
        "net_sale_proceeds",
        "credit_enhancement_proceeds",
        "repurchase_make_whole_proceeds",
        "other_foreclosure_proceeds",
        "non_interest_bearing_upb",
        "principal_forgiveness_upb",
        "repurchase_make_whole_proceeds_flag",
        "foreclosure_principal_write_off_amount",
        "servicing_activity_indicator"
    };
    args.names = dnames;

    const char ** dtype = new const char *[num_cols] {
    		"int64",
    		"date",
    		"category",
    		"float64",
    		"float64",
    		"float64",
    		"float64",
    		"float64",
    		"date",
    		"float64",
    		"category",
    		"category",
    		"category",
    		"date",
    		"date",
    		"date",
    		"date",
    		"float64",
    		"float64",
    		"float64",
    		"float64",
    		"float64",
    		"float64",
    		"float64",
    		"float64",
    		"float64",
    		"float64",
    		"float64",
    		"category",
    		"float64",
    		"category"
        };

        args.dtype = dtype;

		args.input_data_form = gdf_csv_input_form::FILE_PATH;
		args.filepath_or_buffer = (char *)("/tmp/Performance_2000Q1.txt");

	if (  checkFile(args.filepath_or_buffer))
	{
		args.delimiter 		= '|';
		args.lineterminator = '\n';
		args.delim_whitespace = 0;
		args.skipinitialspace = 0;
		args.skiprows 		= 0;
		args.skipfooter 	= 0;
		args.dayfirst 		= 0;
        args.mangle_dupe_cols=true;
        args.num_cols_out=0;

        args.use_cols_int       = NULL;
        args.use_cols_char      = NULL;
        args.use_cols_char_len  = 0;
        args.use_cols_int_len   = 0;


        args.names = NULL;
        args.dtype = NULL;


		error = read_csv(&args);
	}

	EXPECT_TRUE( error == GDF_SUCCESS );
}

TEST(gdf_csv_test, Strings)
{
	const char* fname	= "/tmp/CsvStringsTest.csv";
	const char* names[]	= { "line", "verse" };
	const char* types[]	= { "int32", "str" };

	std::ofstream outfile(fname, std::ofstream::out);
	outfile << names[0] << ',' << names[1] << ',' << '\n';
	outfile << "10,abc def ghi" << '\n';
	outfile << "20,\"jkl mno pqr\"" << '\n';
	outfile << "30,stu \"\"vwx\"\" yz" << '\n';
	outfile.close();
	ASSERT_TRUE( checkFile(fname) );

	{
		csv_read_arg args{};	
		args.input_data_form = gdf_csv_input_form::FILE_PATH;
		args.filepath_or_buffer = fname;
		args.num_cols		= std::extent<decltype(names)>::value;
		args.names			= names;
		args.dtype			= types;
		args.delimiter		= ',';
		args.lineterminator = '\n';
		args.skiprows		= 1;
		args.nrows = -1;
		EXPECT_EQ( read_csv(&args), GDF_SUCCESS );

		// No filtering of any columns
		EXPECT_EQ( args.num_cols_out, args.num_cols );

		// Check the parsed string column metadata
		ASSERT_EQ( args.data[1]->dtype, GDF_STRING );
		auto stringList = reinterpret_cast<NVStrings*>(args.data[1]->data);

		ASSERT_NE( stringList, nullptr );
		auto stringCount = stringList->size();
		ASSERT_EQ( stringCount, 3u );
		auto stringLengths = std::unique_ptr<int[]>{ new int[stringCount] };
		ASSERT_NE( stringList->len(stringLengths.get(), false), 0u );

		// Check the actual strings themselves
		auto strings = std::unique_ptr<char*[]>{ new char*[stringCount] };
		for (size_t i = 0; i < stringCount; ++i) {
			ASSERT_GT( stringLengths[i], 0 );
			strings[i] = new char[stringLengths[i]];
		}
		EXPECT_EQ( stringList->to_host(strings.get(), 0, stringCount), 0 );
		EXPECT_STREQ( strings[0], "abc def ghi" );
		EXPECT_STREQ( strings[1], "\"jkl mno pqr\"" );
		EXPECT_STREQ( strings[2], "stu \"\"vwx\"\" yz" );
		for (size_t i = 0; i < stringCount; ++i) {
			delete[] strings[i];
		}
	}
}

TEST(gdf_csv_test, QuotedStrings)
{
	const char* fname	= "/tmp/CsvQuotedStringsTest.csv";
	const char* names[]	= { "line", "verse" };
	const char* types[]	= { "int32", "str" };

	std::ofstream outfile(fname, std::ofstream::out);
	outfile << names[0] << ',' << names[1] << ',' << '\n';
	outfile << "10,`abc,\ndef, ghi`" << '\n';
	outfile << "20,`jkl, ``mno``, pqr`" << '\n';
	outfile << "30,stu `vwx` yz" << '\n';
	outfile.close();
	ASSERT_TRUE( checkFile(fname) );

	{
		csv_read_arg args{};
		args.input_data_form = gdf_csv_input_form::FILE_PATH;
		args.filepath_or_buffer = fname;
		args.num_cols		= std::extent<decltype(names)>::value;
		args.names			= names;
		args.dtype			= types;
		args.delimiter		= ',';
		args.lineterminator = '\n';
		args.quotechar		= '`';
		args.quoting		= true;	// strip outermost quotechar
		args.doublequote	= true;	// replace double quotechar with single
		args.skiprows		= 1;
		args.nrows = -1;
		EXPECT_EQ( read_csv(&args), GDF_SUCCESS );

		// No filtering of any columns
		EXPECT_EQ( args.num_cols_out, args.num_cols );

		// Check the parsed string column metadata
		ASSERT_EQ( args.data[1]->dtype, GDF_STRING );
		auto stringList = reinterpret_cast<NVStrings*>(args.data[1]->data);

		ASSERT_NE( stringList, nullptr );
		auto stringCount = stringList->size();
		ASSERT_EQ( stringCount, 3u );
		auto stringLengths = std::unique_ptr<int[]>{ new int[stringCount] };
		ASSERT_NE( stringList->len(stringLengths.get(), false), 0u );

		// Check the actual strings themselves
		auto strings = std::unique_ptr<char*[]>{ new char*[stringCount] };
		for (size_t i = 0; i < stringCount; ++i) {
			ASSERT_GT( stringLengths[i], 0 );
			strings[i] = new char[stringLengths[i]];
		}
		EXPECT_EQ( stringList->to_host(strings.get(), 0, stringCount), 0 );
		EXPECT_STREQ( strings[0], "abc,\ndef, ghi" );
		EXPECT_STREQ( strings[1], "jkl, `mno`, pqr" );
		EXPECT_STREQ( strings[2], "stu `vwx` yz" );
		for (size_t i = 0; i < stringCount; ++i) {
			delete[] strings[i];
		}
	}
}

TEST(gdf_csv_test, KeepFullQuotedStrings)
{
	const char* fname	= "/tmp/CsvKeepFullQuotedStringsTest.csv";
	const char* names[]	= { "line", "verse" };
	const char* types[]	= { "int32", "str" };

	std::ofstream outfile(fname, std::ofstream::out);
	outfile << names[0] << ',' << names[1] << ',' << '\n';
	outfile << "10,\"abc,\ndef, ghi\"" << '\n';
	outfile << "20,\"jkl, \"\"mno\"\", pqr\"" << '\n';
	outfile << "30,stu \"vwx\" yz" << '\n';
	outfile.close();
	ASSERT_TRUE( checkFile(fname) );

	{
		csv_read_arg args{};
		args.input_data_form = gdf_csv_input_form::FILE_PATH;
		args.filepath_or_buffer = fname;
		args.num_cols		= std::extent<decltype(names)>::value;
		args.names			= names;
		args.dtype			= types;
		args.delimiter		= ',';
		args.lineterminator = '\n';
		args.quotechar		= '\"';
		args.quoting		= false;	// do not strip outermost quotechar
		args.doublequote	= false;	// do not replace double quotechar with single
		args.skiprows		= 1;
		args.nrows = -1;
		EXPECT_EQ( read_csv(&args), GDF_SUCCESS );

		// No filtering of any columns
		EXPECT_EQ( args.num_cols_out, args.num_cols );

		// Check the parsed string column metadata
		ASSERT_EQ( args.data[1]->dtype, GDF_STRING );
		auto stringList = reinterpret_cast<NVStrings*>(args.data[1]->data);

		ASSERT_NE( stringList, nullptr );
		auto stringCount = stringList->size();
		ASSERT_EQ( stringCount, 3u );
		auto stringLengths = std::unique_ptr<int[]>{ new int[stringCount] };
		ASSERT_NE( stringList->len(stringLengths.get(), false), 0u );

		// Check the actual strings themselves
		auto strings = std::unique_ptr<char*[]>{ new char*[stringCount] };
		for (size_t i = 0; i < stringCount; ++i) {
			ASSERT_GT( stringLengths[i], 0 );
			strings[i] = new char[stringLengths[i]];
		}
		EXPECT_EQ( stringList->to_host(strings.get(), 0, stringCount), 0 );
		EXPECT_STREQ( strings[0], "\"abc,\ndef, ghi\"" );
		EXPECT_STREQ( strings[1], "\"jkl, \"\"mno\"\", pqr\"" );
		EXPECT_STREQ( strings[2], "stu \"vwx\" yz" );
		for (size_t i = 0; i < stringCount; ++i) {
			delete[] strings[i];
		}
	}
}

TEST(gdf_csv_test, SpecifiedBoolValues)
{
	const char* fname			= "/tmp/CsvSpecifiedBoolValuesTest.csv";
	const char* names[]			= { "A", "B", "C" };
	const char* types[]			= { "int32", "int32", "short" };
	const char* trueValues[]	= { "yes", "Yes", "YES", "foo", "FOO" };
	const char* falseValues[]	= { "no", "No", "NO", "Bar", "bar" };

	std::ofstream outfile(fname, std::ofstream::out);
	outfile << "YES,1,bar\nno,2,FOO\nBar,3,yes\nNo,4,NO\nYes,5,foo\n";
	outfile.close();
	ASSERT_TRUE( checkFile(fname) );

	{
		csv_read_arg args{};
		args.input_data_form		= gdf_csv_input_form::FILE_PATH;
		args.filepath_or_buffer		= fname;
		args.num_cols			= std::extent<decltype(names)>::value;
		args.names				= names;
		args.dtype				= types;
		args.delimiter			= ',';
		args.lineterminator 	= '\n';
		args.true_values		= trueValues;
		args.num_true_values	= std::extent<decltype(trueValues)>::value;
		args.false_values		= falseValues;
		args.num_false_values	= std::extent<decltype(falseValues)>::value;
		args.nrows = -1;
		EXPECT_EQ( read_csv(&args), GDF_SUCCESS );

		// Booleans are the same (integer) data type, but valued at 0 or 1
		EXPECT_EQ( args.num_cols_out, args.num_cols );
		ASSERT_EQ( args.data[0]->dtype, GDF_INT32 );
		ASSERT_EQ( args.data[2]->dtype, GDF_INT16 );

		auto firstCol = gdf_host_column<int32_t>(args.data[0]);
		EXPECT_THAT(firstCol.hostdata(), ::testing::ElementsAre(1, 0, 0, 0, 1));
		auto thirdCol = gdf_host_column<int16_t>(args.data[2]);
		EXPECT_THAT(thirdCol.hostdata(), ::testing::ElementsAre(0, 1, 1, 0, 1));
	}
}

TEST(gdf_csv_test, Dates)
{
	const char* fname			= "/tmp/CsvDatesTest.csv";
	const char* names[]			= { "A" };
	const char* types[]			= { "date" };

	std::ofstream outfile(fname, std::ofstream::out);
	outfile << "05/03/2001\n31/10/2010\n20/10/1994\n18/10/1990\n1/1/1970\n";
	outfile << "18/04/1995\n14/07/1994\n07/06/2006\n16/09/2005\n2/2/1970\n";
	outfile.close();
	ASSERT_TRUE( checkFile(fname) );

	{
		csv_read_arg args{};
		args.input_data_form	= gdf_csv_input_form::FILE_PATH;
		args.filepath_or_buffer	= fname;
		args.num_cols			= std::extent<decltype(names)>::value;
		args.names				= names;
		args.dtype				= types;
		args.delimiter			= ',';
		args.lineterminator 	= '\n';
		args.dayfirst			= true;
		args.nrows				= -1;
		EXPECT_EQ( read_csv(&args), GDF_SUCCESS );

		EXPECT_EQ( args.num_cols_out, args.num_cols );
		ASSERT_EQ( args.data[0]->dtype, GDF_DATE64 );

		auto ACol = gdf_host_column<uint64_t>(args.data[0]);
		EXPECT_THAT( ACol.hostdata(),
			::testing::ElementsAre(983750400000, 1288483200000, 782611200000,
								   656208000000, 0, 798163200000, 774144000000,
								   1149638400000, 1126828800000, 2764800000) );
	}
}

TEST(gdf_csv_test, FloatingPoint)
{
	const char* fname			= "/tmp/CsvFloatingPoint.csv";
	const char* names[]			= { "A" };
	const char* types[]			= { "float32" };

	std::ofstream outfile(fname, std::ofstream::out);
	outfile << "5.6;0.5679e2;1.2e10;0.07e1;3000e-3;12.34e0;";
	outfile.close();
	ASSERT_TRUE( checkFile(fname) );

	{
		csv_read_arg args{};
		args.input_data_form    = gdf_csv_input_form::FILE_PATH;
		args.filepath_or_buffer = fname;
		args.num_cols           = std::extent<decltype(names)>::value;
		args.names              = names;
		args.dtype              = types;
		args.decimal            = '.';
		args.delimiter          = ',';
		args.lineterminator     = ';';
		args.nrows              = -1;
		EXPECT_EQ( read_csv(&args), GDF_SUCCESS );

		EXPECT_EQ( args.num_cols_out, args.num_cols );
		ASSERT_EQ( args.data[0]->dtype, GDF_FLOAT32 );

		auto ACol = gdf_host_column<float>(args.data[0]);
		EXPECT_THAT( ACol.hostdata(),
			::testing::Pointwise(FloatNearPointwise(1e-6),
				std::vector<float>{ 5.6, 56.79, 12000000000, 0.7, 3.000, 12.34 }) );
	}
}
