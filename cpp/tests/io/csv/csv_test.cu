/*
 * Copyright (c) 2018, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <cstdlib>
#include <iostream>
#include <fstream>
#include <vector>
#include <sys/stat.h>

#include "gtest/gtest.h"
#include "gmock/gmock.h"

#include <cudf.h>
#include <NVStrings.h>

bool checkFile(const char *fname)
{
	struct stat st;
	return (stat(fname, &st) ? 0 : 1);
}

// DESCRIPTION: Simple test internal helper class to transfer cudf column data
// from device to host for test comparisons and debugging/development
template <typename T>
class gdf_host_column
{
public:
	gdf_host_column() = delete;
	explicit gdf_host_column(gdf_column* const col)
	{
		m_hostdata = std::vector<T>(col->size);
		hipMemcpy(m_hostdata.data(), col->data, sizeof(T) * col->size, hipMemcpyDeviceToHost);
	}

	auto hostdata() const -> const auto&
	{
		return m_hostdata;
	}
	void print() const
	{
		for (size_t i = 0; i < m_hostdata.size(); ++i)
		{
			std::cout << "[" << i << "]: value=" << m_hostdata[i] << "\n";
		}
	}

private:
	std::vector<T> m_hostdata;
};

TEST(gdf_csv_test, Simple)
{
	const char* fname	= "/tmp/CsvSimpleTest.csv";
	const char* names[]	= { "A", "B", "C", "D", "E", "F", "G", "H", "I", "J" };
	const char* types[]	= { "int32", "int32", "int32", "int32", "int32",
							"int32", "int32", "int32", "int32", "int32", };

	std::ofstream outfile(fname, std::ofstream::out);
	outfile <<	"10,20,30,40,50,60,70,80,90,100\n"\
				"11,21,31,41,51,61,71,81,91,101\n"\
				"12,22,32,42,52,62,72,82,92,102\n"\
				"13,23,33,43,53,63,73,83,93,103\n";
	outfile.close();
	ASSERT_TRUE( checkFile(fname) );

	{
		csv_read_arg args{};
		args.input_data_form = gdf_csv_input_form::FILE_PATH;
		args.filepath_or_buffer = fname;
		args.num_cols		= std::extent<decltype(names)>::value;
		args.names			= names;
		args.dtype			= types;
		args.delimiter		= ',';
		args.lineterminator = '\n';
		EXPECT_EQ( read_csv(&args), GDF_SUCCESS );

		auto firstCol = gdf_host_column<int32_t>(args.data[0]);
		auto sixthCol = gdf_host_column<int32_t>(args.data[5]);
		EXPECT_THAT(firstCol.hostdata(), ::testing::ElementsAre(10, 11, 12, 13));
		EXPECT_THAT(sixthCol.hostdata(), ::testing::ElementsAre(60, 61, 62, 63));
	}
}

TEST(gdf_csv_test, MortPerf)
{
	gdf_error error = GDF_SUCCESS;

	csv_read_arg	args;
	const int num_cols = 31;

    args.num_cols = num_cols;

    const char ** dnames = new const char *[num_cols] {
        "loan_id",
        "monthly_reporting_period",
        "servicer",
        "interest_rate",
        "current_actual_upb",
        "loan_age",
        "remaining_months_to_legal_maturity",
        "adj_remaining_months_to_maturity",
        "maturity_date",
        "msa",
        "current_loan_delinquency_status",
        "mod_flag",
        "zero_balance_code",
        "zero_balance_effective_date",
        "last_paid_installment_date",
        "foreclosed_after",
        "disposition_date",
        "foreclosure_costs",
        "prop_preservation_and_repair_costs",
        "asset_recovery_costs",
        "misc_holding_expenses",
        "holding_taxes",
        "net_sale_proceeds",
        "credit_enhancement_proceeds",
        "repurchase_make_whole_proceeds",
        "other_foreclosure_proceeds",
        "non_interest_bearing_upb",
        "principal_forgiveness_upb",
        "repurchase_make_whole_proceeds_flag",
        "foreclosure_principal_write_off_amount",
        "servicing_activity_indicator"
    };
    args.names = dnames;

    const char ** dtype = new const char *[num_cols] {
    		"int64",
    		"date",
    		"category",
    		"float64",
    		"float64",
    		"float64",
    		"float64",
    		"float64",
    		"date",
    		"float64",
    		"category",
    		"category",
    		"category",
    		"date",
    		"date",
    		"date",
    		"date",
    		"float64",
    		"float64",
    		"float64",
    		"float64",
    		"float64",
    		"float64",
    		"float64",
    		"float64",
    		"float64",
    		"float64",
    		"float64",
    		"category",
    		"float64",
    		"category"
        };

        args.dtype = dtype;

		args.input_data_form = gdf_csv_input_form::FILE_PATH;
		args.filepath_or_buffer = (char *)("/tmp/Performance_2000Q1.txt");

	if (  checkFile(args.filepath_or_buffer))
	{
		args.delimiter 		= '|';
		args.lineterminator = '\n';
		args.delim_whitespace = 0;
		args.skipinitialspace = 0;
		args.skiprows 		= 0;
		args.skipfooter 	= 0;
		args.dayfirst 		= 0;
        args.mangle_dupe_cols=true;
        args.num_cols_out=0;

        args.use_cols_int       = NULL;
        args.use_cols_char      = NULL;
        args.use_cols_char_len  = 0;
        args.use_cols_int_len   = 0;


        args.names = NULL;
        args.dtype = NULL;


		error = read_csv(&args);
	}

	EXPECT_TRUE( error == GDF_SUCCESS );
}

TEST(gdf_csv_test, Strings)
{
	const char* fname	= "/tmp/CsvStringsTest.csv";
	const char* names[]	= { "line", "verse" };
	const char* types[]	= { "int32", "str" };

	std::ofstream outfile(fname, std::ofstream::out);
	outfile << names[0] << ',' << names[1] << ',' << '\n';
	outfile << "10,abc def ghi" << '\n';
	outfile << "20,\"jkl mno pqr\"" << '\n';
	outfile << "30,stu \"\"vwx\"\" yz" << '\n';
	outfile.close();
	ASSERT_TRUE( checkFile(fname) );

	{
		csv_read_arg args{};	
		args.input_data_form = gdf_csv_input_form::FILE_PATH;
		args.filepath_or_buffer = fname;
		args.num_cols		= std::extent<decltype(names)>::value;
		args.names			= names;
		args.dtype			= types;
		args.delimiter		= ',';
		args.lineterminator = '\n';
		args.skiprows		= 1;
		EXPECT_EQ( read_csv(&args), GDF_SUCCESS );

		// No filtering of any columns
		EXPECT_EQ( args.num_cols_out, args.num_cols );

		// Check the parsed string column metadata
		ASSERT_EQ( args.data[1]->dtype, GDF_STRING );
		auto stringList = reinterpret_cast<NVStrings*>(args.data[1]->data);

		ASSERT_NE( stringList, nullptr );
		auto stringCount = stringList->size();
		ASSERT_EQ( stringCount, 3u );
		auto stringLengths = std::unique_ptr<int[]>{ new int[stringCount] };
		ASSERT_NE( stringList->len(stringLengths.get(), false), 0u );

		// Check the actual strings themselves
		auto strings = std::unique_ptr<char*[]>{ new char*[stringCount] };
		for (size_t i = 0; i < stringCount; ++i) {
			ASSERT_GT( stringLengths[i], 0 );
			strings[i] = new char[stringLengths[i]];
		}
		EXPECT_EQ( stringList->to_host(strings.get(), 0, stringCount), 0 );
		EXPECT_STREQ( strings[0], "abc def ghi" );
		EXPECT_STREQ( strings[1], "\"jkl mno pqr\"" );
		EXPECT_STREQ( strings[2], "stu \"\"vwx\"\" yz" );
		for (size_t i = 0; i < stringCount; ++i) {
			delete[] strings[i];
		}
	}
}

TEST(gdf_csv_test, QuotedStrings)
{
	const char* fname	= "/tmp/CsvQuotedStringsTest.csv";
	const char* names[]	= { "line", "verse" };
	const char* types[]	= { "int32", "str" };

	std::ofstream outfile(fname, std::ofstream::out);
	outfile << names[0] << ',' << names[1] << ',' << '\n';
	outfile << "10,`abc,\ndef, ghi`" << '\n';
	outfile << "20,`jkl, ``mno``, pqr`" << '\n';
	outfile << "30,stu `vwx` yz" << '\n';
	outfile.close();
	ASSERT_TRUE( checkFile(fname) );

	{
		csv_read_arg args{};
		args.input_data_form = gdf_csv_input_form::FILE_PATH;
		args.filepath_or_buffer = fname;
		args.num_cols		= std::extent<decltype(names)>::value;
		args.names			= names;
		args.dtype			= types;
		args.delimiter		= ',';
		args.lineterminator = '\n';
		args.quotechar		= '`';
		args.quoting		= true;	// strip outermost quotechar
		args.doublequote	= true;	// replace double quotechar with single
		args.skiprows		= 1;
		EXPECT_EQ( read_csv(&args), GDF_SUCCESS );

		// No filtering of any columns
		EXPECT_EQ( args.num_cols_out, args.num_cols );

		// Check the parsed string column metadata
		ASSERT_EQ( args.data[1]->dtype, GDF_STRING );
		auto stringList = reinterpret_cast<NVStrings*>(args.data[1]->data);

		ASSERT_NE( stringList, nullptr );
		auto stringCount = stringList->size();
		ASSERT_EQ( stringCount, 3u );
		auto stringLengths = std::unique_ptr<int[]>{ new int[stringCount] };
		ASSERT_NE( stringList->len(stringLengths.get(), false), 0u );

		// Check the actual strings themselves
		auto strings = std::unique_ptr<char*[]>{ new char*[stringCount] };
		for (size_t i = 0; i < stringCount; ++i) {
			ASSERT_GT( stringLengths[i], 0 );
			strings[i] = new char[stringLengths[i]];
		}
		EXPECT_EQ( stringList->to_host(strings.get(), 0, stringCount), 0 );
		EXPECT_STREQ( strings[0], "abc,\ndef, ghi" );
		EXPECT_STREQ( strings[1], "jkl, `mno`, pqr" );
		EXPECT_STREQ( strings[2], "stu `vwx` yz" );
		for (size_t i = 0; i < stringCount; ++i) {
			delete[] strings[i];
		}
	}
}

TEST(gdf_csv_test, KeepFullQuotedStrings)
{
	const char* fname	= "/tmp/CsvKeepFullQuotedStringsTest.csv";
	const char* names[]	= { "line", "verse" };
	const char* types[]	= { "int32", "str" };

	std::ofstream outfile(fname, std::ofstream::out);
	outfile << names[0] << ',' << names[1] << ',' << '\n';
	outfile << "10,\"abc,\ndef, ghi\"" << '\n';
	outfile << "20,\"jkl, \"\"mno\"\", pqr\"" << '\n';
	outfile << "30,stu \"vwx\" yz" << '\n';
	outfile.close();
	ASSERT_TRUE( checkFile(fname) );

	{
		csv_read_arg args{};
		args.input_data_form = gdf_csv_input_form::FILE_PATH;
		args.filepath_or_buffer = fname;
		args.num_cols		= std::extent<decltype(names)>::value;
		args.names			= names;
		args.dtype			= types;
		args.delimiter		= ',';
		args.lineterminator = '\n';
		args.quotechar		= '\"';
		args.quoting		= false;	// do not strip outermost quotechar
		args.doublequote	= false;	// do not replace double quotechar with single
		args.skiprows		= 1;
		EXPECT_EQ( read_csv(&args), GDF_SUCCESS );

		// No filtering of any columns
		EXPECT_EQ( args.num_cols_out, args.num_cols );

		// Check the parsed string column metadata
		ASSERT_EQ( args.data[1]->dtype, GDF_STRING );
		auto stringList = reinterpret_cast<NVStrings*>(args.data[1]->data);

		ASSERT_NE( stringList, nullptr );
		auto stringCount = stringList->size();
		ASSERT_EQ( stringCount, 3u );
		auto stringLengths = std::unique_ptr<int[]>{ new int[stringCount] };
		ASSERT_NE( stringList->len(stringLengths.get(), false), 0u );

		// Check the actual strings themselves
		auto strings = std::unique_ptr<char*[]>{ new char*[stringCount] };
		for (size_t i = 0; i < stringCount; ++i) {
			ASSERT_GT( stringLengths[i], 0 );
			strings[i] = new char[stringLengths[i]];
		}
		EXPECT_EQ( stringList->to_host(strings.get(), 0, stringCount), 0 );
		EXPECT_STREQ( strings[0], "\"abc,\ndef, ghi\"" );
		EXPECT_STREQ( strings[1], "\"jkl, \"\"mno\"\", pqr\"" );
		EXPECT_STREQ( strings[2], "stu \"vwx\" yz" );
		for (size_t i = 0; i < stringCount; ++i) {
			delete[] strings[i];
		}
	}
}

TEST(gdf_csv_test, SpecifiedBoolValues)
{
	const char* fname			= "/tmp/CsvSpecifiedBoolValuesTest.csv";
	const char* names[]			= { "A", "B", "C" };
	const char* types[]			= { "int32", "int32", "short" };
	const char* trueValues[]	= { "yes", "Yes", "YES", "foo", "FOO" };
	const char* falseValues[]	= { "no", "No", "NO", "Bar", "bar" };

	std::ofstream outfile(fname, std::ofstream::out);
	outfile << "YES,1,bar\nno,2,FOO\nBar,3,yes\nNo,4,NO\nYes,5,foo\n";
	outfile.close();
	ASSERT_TRUE( checkFile(fname) );

	{
		csv_read_arg args{};
		args.input_data_form		= gdf_csv_input_form::FILE_PATH;
		args.filepath_or_buffer		= fname;
		args.num_cols			= std::extent<decltype(names)>::value;
		args.names				= names;
		args.dtype				= types;
		args.delimiter			= ',';
		args.lineterminator 	= '\n';
		args.true_values		= trueValues;
		args.num_true_values	= std::extent<decltype(trueValues)>::value;
		args.false_values		= falseValues;
		args.num_false_values	= std::extent<decltype(falseValues)>::value;
		EXPECT_EQ( read_csv(&args), GDF_SUCCESS );

		// Booleans are the same (integer) data type, but valued at 0 or 1
		EXPECT_EQ( args.num_cols_out, args.num_cols );
		ASSERT_EQ( args.data[0]->dtype, GDF_INT32 );
		ASSERT_EQ( args.data[2]->dtype, GDF_INT16 );

		auto firstCol = gdf_host_column<int32_t>(args.data[0]);
		EXPECT_THAT(firstCol.hostdata(), ::testing::ElementsAre(1, 0, 0, 0, 1));
		auto thirdCol = gdf_host_column<int16_t>(args.data[2]);
		EXPECT_THAT(thirdCol.hostdata(), ::testing::ElementsAre(0, 1, 1, 0, 1));
	}
}
