/*
 * Copyright (c) 2022, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <io/fst/lookup_tables.cuh>
#include <io/utilities/hostdevice_vector.hpp>
#include <tests/io/fst/common.hpp>

#include <cudf_test/base_fixture.hpp>
#include <cudf_test/cudf_gtest.hpp>

#include <cudf/scalar/scalar_factories.hpp>
#include <cudf/strings/repeat_strings.hpp>
#include <cudf/types.hpp>

#include <rmm/hip_stream.hpp>
#include <rmm/cuda_stream_view.hpp>
#include <rmm/device_buffer.hpp>
#include <rmm/device_uvector.hpp>

#include <cstdlib>
#include <vector>

namespace {

//------------------------------------------------------------------------------
// CPU-BASED IMPLEMENTATIONS FOR VERIFICATION
//------------------------------------------------------------------------------
/**
 * @brief CPU-based implementation of a finite-state transducer (FST).
 *
 * @tparam InputItT Forward input iterator type to symbols fed into the FST
 * @tparam StateT Type representing states of the finite-state machine
 * @tparam SymbolGroupLutT Sequence container of symbol groups. Each symbol group is a sequence
 * container to symbols within that group.
 * @tparam TransitionTableT Two-dimensional container type
 * @tparam TransducerTableT Two-dimensional container type
 * @tparam OutputItT Forward output iterator type
 * @tparam IndexOutputItT Forward output iterator type
 * @param[in] begin Forward iterator to the beginning of the symbol sequence
 * @param[in] end Forward iterator to one past the last element of the symbol sequence
 * @param[in] init_state The starting state of the finite-state machine
 * @param[in] symbol_group_lut Sequence container of symbol groups. Each symbol group is a sequence
 * container to symbols within that group. The index of the symbol group containing a symbol being
 * read will be used as symbol_gid of the transition and translation tables.
 * @param[in] transition_table The two-dimensional transition table, i.e.,
 * transition_table[state][symbol_gid] -> new_state
 * @param[in] translation_table The two-dimensional transducer table, i.e.,
 * translation_table[state][symbol_gid] -> range_of_output_symbols
 * @param[out] out_tape A forward output iterator to which the transduced input will be written
 * @param[out] out_index_tape A forward output iterator to which indexes of the symbols that
 * actually caused some output are written to
 * @return A pair of iterators to one past the last element of (1) the transduced output symbol
 * sequence and (2) the indexes of
 */
template <typename InputItT,
          typename StateT,
          typename SymbolGroupLutT,
          typename TransitionTableT,
          typename TransducerTableT,
          typename OutputItT,
          typename IndexOutputItT>
static std::pair<OutputItT, IndexOutputItT> fst_baseline(InputItT begin,
                                                         InputItT end,
                                                         StateT const& init_state,
                                                         SymbolGroupLutT symbol_group_lut,
                                                         TransitionTableT transition_table,
                                                         TransducerTableT translation_table,
                                                         OutputItT out_tape,
                                                         IndexOutputItT out_index_tape)
{
  // Initialize "FSM" with starting state
  StateT state = init_state;

  // To track the symbol offset within the input that caused the FST to output
  std::size_t in_offset = 0;
  for (auto it = begin; it < end; it++) {
    // The symbol currently being read
    auto const& symbol = *it;

    // Iterate over symbol groups and search for the first symbol group containing the current
    // symbol, if no match is found we use cend(symbol_group_lut) as the "catch-all" symbol group
    auto symbol_group_it =
      std::find_if(std::cbegin(symbol_group_lut), std::cend(symbol_group_lut), [symbol](auto& sg) {
        return std::find(std::cbegin(sg), std::cend(sg), symbol) != std::cend(sg);
      });
    auto symbol_group = std::distance(std::cbegin(symbol_group_lut), symbol_group_it);

    // Output the translated symbols to the output tape
    out_tape = std::copy(std::cbegin(translation_table[state][symbol_group]),
                         std::cend(translation_table[state][symbol_group]),
                         out_tape);

    auto out_size = std::distance(std::cbegin(translation_table[state][symbol_group]),
                                  std::cend(translation_table[state][symbol_group]));

    out_index_tape = std::fill_n(out_index_tape, out_size, in_offset);

    // Transition the state of the finite-state machine
    state = static_cast<char>(transition_table[state][symbol_group]);

    // Continue with next symbol from input tape
    in_offset++;
  }
  return {out_tape, out_index_tape};
}

using namespace cudf::test::io::json;
}  // namespace

// Base test fixture for tests
struct FstTest : public cudf::test::BaseFixture {
};

TEST_F(FstTest, GroundTruth)
{
  // Type used to represent the atomic symbol type used within the finite-state machine
  using SymbolT = char;

  // Type sufficiently large to index symbols within the input and output (may be unsigned)
  using SymbolOffsetT = uint32_t;

  // Helper class to set up transition table, symbol group lookup table, and translation table
  using DfaFstT = cudf::io::fst::detail::Dfa<char, NUM_SYMBOL_GROUPS, TT_NUM_STATES>;

  // Prepare cuda stream for data transfers & kernels
  rmm::hip_stream stream{};
  rmm::cuda_stream_view stream_view(stream);

  // Test input
  std::string input = R"(  {)"
                      R"("category": "reference",)"
                      R"("index:" [4,12,42],)"
                      R"("author": "Nigel Rees",)"
                      R"("title": "Sayings of the Century",)"
                      R"("price": 8.95)"
                      R"(}  )"
                      R"({)"
                      R"("category": "reference",)"
                      R"("index:" [4,{},null,{"a":[]}],)"
                      R"("author": "Nigel Rees",)"
                      R"("title": "Sayings of the Century",)"
                      R"("price": 8.95)"
                      R"(}  {} [] [ ])";

  size_t string_size                 = input.size() * (1 << 10);
  auto d_input_scalar                = cudf::make_string_scalar(input);
  auto& d_string_scalar              = static_cast<cudf::string_scalar&>(*d_input_scalar);
  const cudf::size_type repeat_times = string_size / input.size();
  auto d_input_string                = cudf::strings::repeat_string(d_string_scalar, repeat_times);
  auto& d_input = static_cast<cudf::scalar_type_t<std::string>&>(*d_input_string);
  input         = d_input.to_string(stream);

  // Prepare input & output buffers
  constexpr std::size_t single_item = 1;
  hostdevice_vector<SymbolT> output_gpu(input.size(), stream_view);
  hostdevice_vector<SymbolOffsetT> output_gpu_size(single_item, stream_view);
  hostdevice_vector<SymbolOffsetT> out_indexes_gpu(input.size(), stream_view);

  // Run algorithm
  DfaFstT parser{pda_sgs, pda_state_tt, pda_out_tt, stream.value()};

  // Allocate device-side temporary storage & run algorithm
  parser.Transduce(d_input.data(),
                   static_cast<SymbolOffsetT>(d_input.size()),
                   output_gpu.device_ptr(),
                   out_indexes_gpu.device_ptr(),
                   output_gpu_size.device_ptr(),
                   start_state,
                   stream.value());

  // Async copy results from device to host
  output_gpu.device_to_host(stream.view());
  out_indexes_gpu.device_to_host(stream.view());
  output_gpu_size.device_to_host(stream.view());

  // Prepare CPU-side results for verification
  std::string output_cpu{};
  std::vector<SymbolOffsetT> out_index_cpu{};
  output_cpu.reserve(input.size());
  out_index_cpu.reserve(input.size());

  // Run CPU-side algorithm
  fst_baseline(std::begin(input),
               std::end(input),
               start_state,
               pda_sgs,
               pda_state_tt,
               pda_out_tt,
               std::back_inserter(output_cpu),
               std::back_inserter(out_index_cpu));

  // Make sure results have been copied back to host
  stream.synchronize();

  // Verify results
  ASSERT_EQ(output_gpu_size[0], output_cpu.size());
  CUDF_TEST_EXPECT_VECTOR_EQUAL(output_gpu, output_cpu, output_cpu.size());
  CUDF_TEST_EXPECT_VECTOR_EQUAL(out_indexes_gpu, out_index_cpu, output_cpu.size());
}

CUDF_TEST_PROGRAM_MAIN()
