/*
 * Copyright (c) 2019, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <algorithm>
#include <cmath>
#include <ctgmath>
#include <cudf/copying.hpp>
#include <cudf/stream_compaction.hpp>
#include <cudf/table/table.hpp>
#include <cudf/table/table_view.hpp>
#include <cudf/types.hpp>
#include <tests/utilities/base_fixture.hpp>
#include <tests/utilities/column_utilities.hpp>
#include <tests/utilities/column_wrapper.hpp>
#include <tests/utilities/table_utilities.hpp>
#include <tests/utilities/type_lists.hpp>

template <typename T>
struct UniqueCountCommon : public cudf::test::BaseFixture {
};

TYPED_TEST_CASE(UniqueCountCommon, cudf::test::NumericTypes);

TYPED_TEST(UniqueCountCommon, NoNull)
{
  using T = TypeParam;

  std::vector<T> input = cudf::test::make_type_param_vector<T>(
    {1, 3, 3, 4, 31, 1, 8, 2, 0, 4, 1, 4, 10, 40, 31, 42, 0, 42, 8, 5, 4});

  cudf::test::fixed_width_column_wrapper<T> input_col(input.begin(), input.end());

  cudf::size_type expected = std::set<double>(input.begin(), input.end()).size();
  EXPECT_EQ(expected, cudf::experimental::unique_count(input_col, false, false));
}

struct UniqueCount : public cudf::test::BaseFixture {
};

TEST_F(UniqueCount, WithNull)
{
  using T = int32_t;

  // Considering 70 as null
  std::vector<T> input = {1, 3, 3, 70, 31, 1, 8, 2, 0, 70, 1, 70, 10, 40, 31, 42, 0, 42, 8, 5, 70};
  std::vector<cudf::size_type> valid = {1, 1, 1, 0, 1, 1, 1, 1, 1, 0, 1,
                                        0, 1, 1, 1, 1, 1, 1, 1, 1, 0};

  cudf::test::fixed_width_column_wrapper<T> input_col(input.begin(), input.end(), valid.begin());

  cudf::size_type expected = std::set<double>(input.begin(), input.end()).size();
  EXPECT_EQ(expected, cudf::experimental::unique_count(input_col, false, false));
}

TEST_F(UniqueCount, IgnoringNull)
{
  using T = int32_t;

  // Considering 70 and 3 as null
  std::vector<T> input = {1, 3, 3, 70, 31, 1, 8, 2, 0, 70, 1, 70, 10, 40, 31, 42, 0, 42, 8, 5, 70};
  std::vector<cudf::size_type> valid = {1, 0, 0, 0, 1, 1, 1, 1, 1, 0, 1,
                                        0, 1, 1, 1, 1, 1, 1, 1, 1, 0};

  cudf::test::fixed_width_column_wrapper<T> input_col(input.begin(), input.end(), valid.begin());

  cudf::size_type expected = std::set<T>(input.begin(), input.end()).size();
  // Removing 2 from expected to remove count for 70 and 3
  EXPECT_EQ(expected - 2, cudf::experimental::unique_count(input_col, true, false));
}

TEST_F(UniqueCount, WithNansAndNull)
{
  using T = float;

  std::vector<T> input               = {1,  3,  NAN, 70, 31,  1, 8,   2, 0, 70, 1,
                          70, 10, 40,  31, NAN, 0, NAN, 8, 5, 70};
  std::vector<cudf::size_type> valid = {1, 0, 0, 0, 1, 1, 1, 1, 1, 0, 1,
                                        0, 1, 1, 1, 1, 1, 1, 1, 1, 0};

  cudf::test::fixed_width_column_wrapper<T> input_col{input.begin(), input.end(), valid.begin()};

  cudf::size_type expected = std::set<T>(input.begin(), input.end()).size();
  EXPECT_EQ(expected, cudf::experimental::unique_count(input_col, false, false));
}

TEST_F(UniqueCount, WithNansOnly)
{
  using T = float;

  std::vector<T> input               = {1, 3, NAN, 70, 31};
  std::vector<cudf::size_type> valid = {1, 1, 1, 1, 1};

  cudf::test::fixed_width_column_wrapper<T> input_col{input.begin(), input.end(), valid.begin()};

  cudf::size_type expected = 5;
  EXPECT_EQ(expected, cudf::experimental::unique_count(input_col, false, false));
}

TEST_F(UniqueCount, NansAsNullWithNoNull)
{
  using T = float;

  std::vector<T> input               = {1, 3, NAN, 70, 31};
  std::vector<cudf::size_type> valid = {1, 1, 1, 1, 1};

  cudf::test::fixed_width_column_wrapper<T> input_col{input.begin(), input.end(), valid.begin()};

  cudf::size_type expected = 5;
  EXPECT_EQ(expected, cudf::experimental::unique_count(input_col, false, true));
}

TEST_F(UniqueCount, NansAsNullWithNull)
{
  using T = float;

  std::vector<T> input               = {1, 3, NAN, 70, 31};
  std::vector<cudf::size_type> valid = {1, 1, 1, 0, 1};

  cudf::test::fixed_width_column_wrapper<T> input_col{input.begin(), input.end(), valid.begin()};

  cudf::size_type expected = 4;
  EXPECT_EQ(expected, cudf::experimental::unique_count(input_col, false, true));
}

TEST_F(UniqueCount, NansAsNullWithIgnoreNull)
{
  using T = float;

  std::vector<T> input               = {1, 3, NAN, 70, 31};
  std::vector<cudf::size_type> valid = {1, 1, 1, 0, 1};

  cudf::test::fixed_width_column_wrapper<T> input_col{input.begin(), input.end(), valid.begin()};

  cudf::size_type expected = 3;
  EXPECT_EQ(expected, cudf::experimental::unique_count(input_col, true, true));
}

TEST_F(UniqueCount, EmptyColumn)
{
  using T = float;

  cudf::test::fixed_width_column_wrapper<T> input_col{};

  cudf::size_type expected = 0;
  EXPECT_EQ(expected, cudf::experimental::unique_count(input_col, true, true));
}

TEST_F(UniqueCount, StringColumnWithNull)
{
  cudf::test::strings_column_wrapper input_col{
    {"", "this", "is", "this", "This", "a", "column", "of", "the", "strings"},
    {1, 1, 1, 1, 1, 1, 1, 1, 0, 1}};

  cudf::size_type expected =
    (std::vector<std::string>{"", "this", "is", "This", "a", "column", "of", "strings"}).size();
  EXPECT_EQ(expected, cudf::experimental::unique_count(input_col, true, false));
}

struct DropDuplicate : public cudf::test::BaseFixture {
};

TEST_F(DropDuplicate, NonNullTable)
{
  cudf::test::fixed_width_column_wrapper<int32_t> col1{{5, 4, 3, 5, 8, 5}};
  cudf::test::fixed_width_column_wrapper<float> col2{{4, 5, 3, 4, 9, 4}};
  cudf::test::fixed_width_column_wrapper<int32_t> col1_key{{20, 20, 20, 19, 21, 9}};
  cudf::test::fixed_width_column_wrapper<int32_t> col2_key{{19, 19, 20, 20, 9, 21}};

  cudf::table_view input{{col1, col2, col1_key, col2_key}};
  std::vector<cudf::size_type> keys{2, 3};

  // Keep first of duplicate
  // The expected table would be sorted in ascending order with respect to keys
  cudf::test::fixed_width_column_wrapper<int32_t> exp_col1_first{{5, 5, 5, 3, 8}};
  cudf::test::fixed_width_column_wrapper<float> exp_col2_first{{4, 4, 4, 3, 9}};
  cudf::test::fixed_width_column_wrapper<int32_t> exp_col1_key_first{{9, 19, 20, 20, 21}};
  cudf::test::fixed_width_column_wrapper<int32_t> exp_col2_key_first{{21, 20, 19, 20, 9}};
  cudf::table_view expected_first{
    {exp_col1_first, exp_col2_first, exp_col1_key_first, exp_col2_key_first}};

  auto got_first =
    drop_duplicates(input, keys, cudf::experimental::duplicate_keep_option::KEEP_FIRST);

  cudf::test::expect_tables_equal(expected_first, got_first->view());

  // keep last of duplicate
  cudf::test::fixed_width_column_wrapper<int32_t> exp_col1_last{{5, 5, 4, 3, 8}};
  cudf::test::fixed_width_column_wrapper<float> exp_col2_last{{4, 4, 5, 3, 9}};
  cudf::test::fixed_width_column_wrapper<int32_t> exp_col1_key_last{{9, 19, 20, 20, 21}};
  cudf::test::fixed_width_column_wrapper<int32_t> exp_col2_key_last{{21, 20, 19, 20, 9}};
  cudf::table_view expected_last{
    {exp_col1_last, exp_col2_last, exp_col1_key_last, exp_col2_key_last}};

  auto got_last =
    drop_duplicates(input, keys, cudf::experimental::duplicate_keep_option::KEEP_LAST);

  cudf::test::expect_tables_equal(expected_last, got_last->view());

  // Keep unique
  cudf::test::fixed_width_column_wrapper<int32_t> exp_col1_unique{{5, 5, 3, 8}};
  cudf::test::fixed_width_column_wrapper<float> exp_col2_unique{{4, 4, 3, 9}};
  cudf::test::fixed_width_column_wrapper<int32_t> exp_col1_key_unique{{9, 19, 20, 21}};
  cudf::test::fixed_width_column_wrapper<int32_t> exp_col2_key_unique{{21, 20, 20, 9}};
  cudf::table_view expected_unique{
    {exp_col1_unique, exp_col2_unique, exp_col1_key_unique, exp_col2_key_unique}};

  auto got_unique =
    drop_duplicates(input, keys, cudf::experimental::duplicate_keep_option::KEEP_NONE);

  cudf::test::expect_tables_equal(expected_unique, got_unique->view());
}

TEST_F(DropDuplicate, WithNull)
{
  cudf::test::fixed_width_column_wrapper<int32_t> col{{5, 4, 3, 5, 8, 1}, {1, 0, 1, 1, 1, 1}};
  cudf::test::fixed_width_column_wrapper<int32_t> key{{20, 20, 20, 19, 21, 19}, {1, 0, 0, 1, 1, 1}};
  cudf::table_view input{{col, key}};
  std::vector<cudf::size_type> keys{1};

  // Keep first of duplicate
  cudf::test::fixed_width_column_wrapper<int32_t> exp_col_first{{4, 5, 5, 8}, {0, 1, 1, 1}};
  cudf::test::fixed_width_column_wrapper<int32_t> exp_key_col_first{{20, 19, 20, 21}, {0, 1, 1, 1}};
  cudf::table_view expected_first{{exp_col_first, exp_key_col_first}};
  auto got_first =
    drop_duplicates(input, keys, cudf::experimental::duplicate_keep_option::KEEP_FIRST, true);

  cudf::test::expect_tables_equal(expected_first, got_first->view());

  // Keep last of duplicate
  cudf::test::fixed_width_column_wrapper<int32_t> exp_col_last{{3, 1, 5, 8}, {1, 1, 1, 1}};
  cudf::test::fixed_width_column_wrapper<int32_t> exp_key_col_last{{20, 19, 20, 21}, {0, 1, 1, 1}};
  cudf::table_view expected_last{{exp_col_last, exp_key_col_last}};
  auto got_last =
    drop_duplicates(input, keys, cudf::experimental::duplicate_keep_option::KEEP_LAST);

  cudf::test::expect_tables_equal(expected_last, got_last->view());

  // Keep unique of duplicate
  cudf::test::fixed_width_column_wrapper<int32_t> exp_col_unique{{5, 8}, {1, 1}};
  cudf::test::fixed_width_column_wrapper<int32_t> exp_key_col_unique{{20, 21}, {1, 1}};
  cudf::table_view expected_unique{{exp_col_unique, exp_key_col_unique}};
  auto got_unique =
    drop_duplicates(input, keys, cudf::experimental::duplicate_keep_option::KEEP_NONE);

  cudf::test::expect_tables_equal(expected_unique, got_unique->view());
}

TEST_F(DropDuplicate, StringKeyColumn)
{
  cudf::test::fixed_width_column_wrapper<int32_t> col{{5, 4, 3, 5, 8, 1}, {1, 0, 1, 1, 1, 1}};
  cudf::test::strings_column_wrapper key_col{{"all", "new", "all", "new", "the", "strings"},
                                             {1, 1, 1, 0, 1, 1}};
  cudf::table_view input{{col, key_col}};
  std::vector<cudf::size_type> keys{1};
  cudf::test::fixed_width_column_wrapper<int32_t> exp_col_last{{5, 3, 4, 1, 8}, {1, 1, 0, 1, 1}};
  cudf::test::strings_column_wrapper exp_key_col_last{{"new", "all", "new", "strings", "the"},
                                                      {0, 1, 1, 1, 1}};
  cudf::table_view expected_last{{exp_col_last, exp_key_col_last}};

  auto got_last =
    drop_duplicates(input, keys, cudf::experimental::duplicate_keep_option::KEEP_LAST);

  cudf::test::expect_tables_equal(expected_last, got_last->view());
}

TEST_F(DropDuplicate, EmptyInputTable)
{
  cudf::test::fixed_width_column_wrapper<int32_t> col(std::initializer_list<int32_t>{});
  cudf::table_view input{{col}};
  std::vector<cudf::size_type> keys{1, 2};

  auto got =
    drop_duplicates(input, keys, cudf::experimental::duplicate_keep_option::KEEP_FIRST, true);

  cudf::test::expect_tables_equal(input, got->view());
}

TEST_F(DropDuplicate, NoColumnInputTable)
{
  cudf::table_view input{std::vector<cudf::column_view>()};
  std::vector<cudf::size_type> keys{1, 2};

  auto got =
    drop_duplicates(input, keys, cudf::experimental::duplicate_keep_option::KEEP_FIRST, true);

  cudf::test::expect_tables_equal(input, got->view());
}

TEST_F(DropDuplicate, EmptyKeys)
{
  cudf::test::fixed_width_column_wrapper<int32_t> col{{5, 4, 3, 5, 8, 1}, {1, 0, 1, 1, 1, 1}};
  cudf::test::fixed_width_column_wrapper<int32_t> empty_col{};
  cudf::table_view input{{col}};
  std::vector<cudf::size_type> keys{};

  auto got =
    drop_duplicates(input, keys, cudf::experimental::duplicate_keep_option::KEEP_FIRST, true);

  cudf::test::expect_tables_equal(cudf::table_view{{empty_col}}, got->view());
}
