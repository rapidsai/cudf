#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <cudf/legacy/stream_compaction.hpp>
#include <cudf/legacy/table.hpp>
#include <cudf/legacy/copying.hpp>

#include <cudf/utilities/error.hpp>

#include <tests/utilities/legacy/column_wrapper.cuh>
#include <tests/utilities/legacy/cudf_test_fixtures.h>
#include <tests/utilities/legacy/cudf_test_utils.cuh>

#include<limits>


template <typename T>
struct DropDuplicatesTest : GdfTest { };

using test_types2 =
  ::testing::Types<int8_t, int16_t, int32_t, int64_t, float, double>;

TYPED_TEST_CASE(DropDuplicatesTest, test_types2);

template <typename T>
void TypedDropDuplicatesTest(cudf::test::column_wrapper<T> source,
                     cudf::test::column_wrapper<T> expected)
{
  gdf_column* inrow[]{source.get()};
  cudf::table input_table(inrow, 1);
  enum cudf::duplicate_keep_option keep=cudf::duplicate_keep_option::KEEP_LAST;
  cudf::table out_table;

  EXPECT_NO_THROW(out_table = cudf::drop_duplicates(input_table, input_table, keep));

  gdf_column result = *(out_table.get_column(0));
  EXPECT_TRUE(expected == result);

  //*
  if (!(expected == result)) {
    std::cout << "expected["<< expected.get()->size<< "]\n";
    expected.print();
    std::cout << "result["<< result.size << "]\n";
    print_gdf_column(&result);
  }
  //*/

  gdf_column_free(&result);
}

constexpr cudf::size_type column_size{1000000};

TYPED_TEST(DropDuplicatesTest, Empty)
{
  TypedDropDuplicatesTest<TypeParam>(
    cudf::test::column_wrapper<TypeParam>{}, //{0, false},
    cudf::test::column_wrapper<TypeParam>{}); //{0, false});
}

TYPED_TEST(DropDuplicatesTest, Distinct)
{
  constexpr cudf::size_type column_size = 
    std::numeric_limits<TypeParam>::max() >1000000? 1000000:
    std::numeric_limits<TypeParam>::max();
  TypedDropDuplicatesTest<TypeParam>(
    cudf::test::column_wrapper<TypeParam>{column_size,
      [](cudf::size_type row) { return row; }, false},
    cudf::test::column_wrapper<TypeParam>{column_size,
      [](cudf::size_type row) { return row; },
      [](cudf::size_type row) { return true; }
      });
}

TYPED_TEST(DropDuplicatesTest, SingleValue)
{
  TypedDropDuplicatesTest<TypeParam>(
    cudf::test::column_wrapper<TypeParam>{column_size,
      [](cudf::size_type row) { return 2; }, false},
    cudf::test::column_wrapper<TypeParam>{1,
      [](cudf::size_type row) { return 2; }, 
      [](cudf::size_type row) { return true; }
      });
      //[](cudf::size_type row) { return true; }});
}

TYPED_TEST(DropDuplicatesTest, Duplicate)
{
  TypedDropDuplicatesTest<TypeParam>(
      cudf::test::column_wrapper<TypeParam>{column_size,
      [](cudf::size_type row) { return row%100; }, false},
      cudf::test::column_wrapper<TypeParam>{100,
      [](cudf::size_type row) { return row;  }, 
      [](cudf::size_type row) { return true; }
      });
}


template <class T>
struct DropDuplicatesDoubleTest : GdfTest { };

//using TestingTypes = ::testing::Types<int8_t, int16_t, int32_t, int64_t, float,
//                                      double, cudf::date32, cudf::date64,
//                                      cudf::timestamp, cudf::category>;
//                                   cudf::nvstring_category, cudf::bool8>;

template <typename A, typename B>
struct TypeDefinitions
{
  typedef A Type0;
  typedef B Type1;
};

// The list of types we want to test.
typedef ::testing::Types <TypeDefinitions<int32_t,int32_t>,
                          TypeDefinitions<int32_t,float>,
                          TypeDefinitions<float,int32_t>,
                          TypeDefinitions<int32_t,double>,
                          TypeDefinitions<double, cudf::date32>,
                          TypeDefinitions<cudf::date32, cudf::date64>
                                  > Implementations;

TYPED_TEST_CASE(DropDuplicatesDoubleTest, Implementations);

template <class T>
bool compare_columns_indexed(
    cudf::table out_table,
    cudf::test::column_wrapper<typename T::Type0> expected_col1, 
    cudf::test::column_wrapper<typename T::Type1> expected_col2
    )
{
  int rows = expected_col1.size();
  gdf_column index_col = *(out_table.get_column(0));
  cudf::size_type* index_ptr = ((cudf::size_type*)index_col.data);

  rmm::device_vector<cudf::size_type> ordered(rows);
  thrust::sequence(rmm::exec_policy(0)->on(0), ordered.begin(), ordered.end());
  thrust::sort_by_key(rmm::exec_policy(0)->on(0), index_ptr , index_ptr + rows, ordered.data().get());

  cudf::table destination_table(rows,
                                cudf::column_dtypes(out_table),
                                cudf::column_dtype_infos(out_table), true);
  cudf::gather(&out_table, ordered.data().get(), &destination_table);

  gdf_column result_col1 = *(destination_table.get_column(1));
  EXPECT_TRUE(expected_col1 == result_col1);

  gdf_column result_col2 = *(destination_table.get_column(2));
  EXPECT_TRUE(expected_col2 == result_col2);

  /*
  if (!(expected_col1 == result_col1)) {
    std::cout << "expected_col1["<< expected_col1.get()->size<< "]\n";
    expected_col1.print();
    std::cout << "result_col1["<< result_col1.size << "]\n";
    print_gdf_column(&result_col1);
  }
  if (!(expected_col2 == result_col2)) {
    std::cout << "expected_col2["<< expected_col2.get()->size<< "]\n";
    expected_col2.print();
    std::cout << "result_col2["<< result_col2.size << "]\n";
    print_gdf_column(&result_col2);
  }
  */

  gdf_column_free(&index_col);
  gdf_column_free(&result_col1);
  gdf_column_free(&result_col2);
  return true;
}

template <class T>
void TypedDropDuplicatesTest(
    cudf::test::column_wrapper<typename T::Type0> source_col1, 
    cudf::test::column_wrapper<typename T::Type1> source_col2, 
    cudf::test::column_wrapper<typename T::Type0> expected_col1, 
    cudf::test::column_wrapper<typename T::Type1> expected_col2,
    enum cudf::duplicate_keep_option keep)
{
  cudf::test::column_wrapper<cudf::size_type> index{source_col1.size(),
      [](cudf::size_type row) { return row; },
      [](cudf::size_type row) { return true; }};
   
  gdf_column* inrow[]{index.get(), source_col1.get(), source_col2.get()};
  cudf::table input_table(inrow, 3);
  cudf::table keycol_table(inrow+1, 2);
  cudf::table out_table;

  EXPECT_NO_THROW(out_table = cudf::drop_duplicates(input_table, keycol_table, keep));

  //reorder and compare columns
  compare_columns_indexed<T>(out_table, expected_col1, expected_col2);
}

auto lamda_valid = [](cudf::size_type row) { return true; };

TYPED_TEST(DropDuplicatesDoubleTest, Empty)
{
  TypedDropDuplicatesTest<TypeParam>(
    cudf::test::column_wrapper<typename TypeParam::Type0>{}, //{0, false},
    cudf::test::column_wrapper<typename TypeParam::Type1>{}, //{0, false},
    cudf::test::column_wrapper<typename TypeParam::Type0>{}, //{0, false},
    cudf::test::column_wrapper<typename TypeParam::Type1>{}, //{0, false},
    cudf::duplicate_keep_option::KEEP_LAST);
}

TYPED_TEST(DropDuplicatesDoubleTest, Distinct)
{
  auto lamda_type0 = [](cudf::size_type row) { return typename TypeParam::Type0(row); };
  auto lamda_type1 = [](cudf::size_type row) { return typename TypeParam::Type1(row); };
  cudf::test::column_wrapper<typename TypeParam::Type0> col1{column_size, lamda_type0};
  cudf::test::column_wrapper<typename TypeParam::Type1> col2{column_size, lamda_type1};
  TypedDropDuplicatesTest<TypeParam>(
    col1,
    col2,
    cudf::test::column_wrapper<typename TypeParam::Type0>{column_size, lamda_type0, lamda_valid},
    cudf::test::column_wrapper<typename TypeParam::Type1>{column_size, lamda_type1, lamda_valid},
    cudf::duplicate_keep_option::KEEP_FIRST);

  TypedDropDuplicatesTest<TypeParam>(
    col1,
    col2,
    cudf::test::column_wrapper<typename TypeParam::Type0>{column_size, lamda_type0, lamda_valid},
    cudf::test::column_wrapper<typename TypeParam::Type1>{column_size, lamda_type1, lamda_valid},
    cudf::duplicate_keep_option::KEEP_LAST);

  TypedDropDuplicatesTest<TypeParam>(
    col1,
    col2,
    cudf::test::column_wrapper<typename TypeParam::Type0>{column_size, lamda_type0, lamda_valid},
    cudf::test::column_wrapper<typename TypeParam::Type1>{column_size, lamda_type1, lamda_valid},
    cudf::duplicate_keep_option::KEEP_NONE);
}

TYPED_TEST(DropDuplicatesDoubleTest, Duplicate)
{
  auto lamda_type0 = [](cudf::size_type row) { return typename TypeParam::Type0(row%100); };
  auto lamda_type1 = [](cudf::size_type row) { return typename TypeParam::Type1(row%7); };
  cudf::test::column_wrapper<typename TypeParam::Type0> col1{column_size, lamda_type0};
  cudf::test::column_wrapper<typename TypeParam::Type1> col2{column_size, lamda_type1};
  TypedDropDuplicatesTest<TypeParam>(
    col1,
    col2,
    cudf::test::column_wrapper<typename TypeParam::Type0>{700, lamda_type0, lamda_valid},
    cudf::test::column_wrapper<typename TypeParam::Type1>{700, lamda_type1, lamda_valid},
    cudf::duplicate_keep_option::KEEP_FIRST);

  TypedDropDuplicatesTest<TypeParam>(
    col1,
    col2,
    cudf::test::column_wrapper<typename TypeParam::Type0>{700, lamda_type0, lamda_valid},
    cudf::test::column_wrapper<typename TypeParam::Type1>{700,
      [](cudf::size_type row) { return typename TypeParam::Type1((column_size-700+row)%7); }, lamda_valid},
    cudf::duplicate_keep_option::KEEP_LAST);

  TypedDropDuplicatesTest<TypeParam>(
    col1,
    col2,
    cudf::test::column_wrapper<typename TypeParam::Type0>{0, lamda_type0, lamda_valid},
    cudf::test::column_wrapper<typename TypeParam::Type1>{0, lamda_type1, lamda_valid},
    cudf::duplicate_keep_option::KEEP_NONE);
}

