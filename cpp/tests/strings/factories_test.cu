/*
 * Copyright (c) 2019, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <cudf/column/column_factories.hpp>
#include <cudf/strings/strings_column_view.hpp>
#include <cudf/types.hpp>
#include <tests/utilities/base_fixture.hpp>

#include <gmock/gmock.h>

#include <vector>
#include <cstring>


struct FactoriesTest : public cudf::test::BaseFixture {};

TEST_F(FactoriesTest, CreateColumnFromArray)
{
    std::vector<const char*> h_test_strings{ "the quick brown fox jumps over the lazy dog",
                                             "the fat cat lays next to the other accénted cat",
                                             "a slow moving turtlé cannot catch the bird",
                                             "which can be composéd together to form a more complete",
                                             "thé result does not include the value in the sum in",
                                             "", nullptr, "absent stop words" };

    cudf::size_type memsize = 0;
    for( auto itr=h_test_strings.begin(); itr!=h_test_strings.end(); ++itr )
        memsize += *itr ? (cudf::size_type)strlen(*itr) : 0;
    cudf::size_type count = (cudf::size_type)h_test_strings.size();
    thrust::host_vector<char> h_buffer(memsize);
    thrust::device_vector<char> d_buffer(memsize);
    thrust::host_vector<thrust::pair<const char*,cudf::size_type> > strings(count);
    thrust::host_vector<cudf::size_type> h_offsets(count+1);
    cudf::size_type offset = 0;
    cudf::size_type nulls = 0;
    h_offsets[0] = 0;
    for( cudf::size_type idx=0; idx < count; ++idx )
    {
        const char* str = h_test_strings[idx];
        if( !str )
        {
            strings[idx] = thrust::pair<const char*,cudf::size_type>{nullptr,0};
            nulls++;
        }
        else
        {
            cudf::size_type length = (cudf::size_type)strlen(str);
            memcpy( h_buffer.data() + offset, str, length );
            strings[idx] = thrust::pair<const char*,cudf::size_type>{d_buffer.data().get()+offset,length};
            offset += length;
        }
        h_offsets[idx+1] = offset;
    }
    rmm::device_vector<thrust::pair<const char*,cudf::size_type>> d_strings(strings);
    hipMemcpy( d_buffer.data().get(), h_buffer.data(), memsize, hipMemcpyHostToDevice );
    auto column = cudf::make_strings_column( d_strings );
    EXPECT_EQ(column->type(), cudf::data_type{cudf::STRING});
    EXPECT_EQ(column->null_count(), nulls);
    if( nulls )
    {
        EXPECT_TRUE(column->nullable());
        EXPECT_TRUE(column->has_nulls());
    }
    EXPECT_EQ(2, column->num_children());

    cudf::strings_column_view strings_view(column->view());
    EXPECT_EQ( strings_view.size(), count);
    EXPECT_EQ( strings_view.offsets().size(), count+1 );
    EXPECT_EQ( strings_view.chars().size(), memsize );

    // check string data
    auto strings_data = cudf::strings::create_offsets(strings_view);
    thrust::host_vector<char> h_chars_data(strings_data.first);
    thrust::host_vector<cudf::size_type> h_offsets_data(strings_data.second);
    EXPECT_EQ( memcmp(h_buffer.data(), h_chars_data.data(), h_buffer.size()), 0 );
    EXPECT_EQ( memcmp(h_offsets.data(), h_offsets_data.data(), h_offsets.size()*sizeof(cudf::size_type)), 0);
}

TEST_F(FactoriesTest, CreateColumnFromOffsets)
{
    std::vector<const char*> h_test_strings{ "the quick brown fox jumps over the lazy dog",
                                         "the fat cat lays next to the other accénted cat",
                                         "a slow moving turtlé cannot catch the bird",
                                         "which can be composéd together to form a more complete",
                                         "thé result does not include the value in the sum in",
                                         "absent stop words" };

    cudf::size_type memsize = 0;
    for( auto itr=h_test_strings.begin(); itr!=h_test_strings.end(); ++itr )
        memsize += *itr ? (cudf::size_type)strlen(*itr) : 0;
    cudf::size_type count = (cudf::size_type)h_test_strings.size();
    thrust::host_vector<char> h_buffer(memsize);
    thrust::host_vector<cudf::size_type> h_offsets(count+1);
    cudf::size_type offset = 0;
    h_offsets[0] = 0;
    for( cudf::size_type idx=0; idx < count; ++idx )
    {
        const char* str = h_test_strings[idx];
        if( str )
        {
            cudf::size_type length = (cudf::size_type)strlen(str);
            memcpy( h_buffer.data() + offset, str, length );
            offset += length;
        }
        h_offsets[idx+1] = offset;
    }
    rmm::device_vector<char> d_buffer(h_buffer);
    rmm::device_vector<cudf::size_type> d_offsets(h_offsets);
    rmm::device_vector<cudf::bitmask_type> d_nulls;
    auto column = cudf::make_strings_column( d_buffer, d_offsets, d_nulls, 0 );
    EXPECT_EQ(column->type(), cudf::data_type{cudf::STRING});
    EXPECT_EQ(column->null_count(), 0);
    EXPECT_EQ(2, column->num_children());

    cudf::strings_column_view strings_view(column->view());
    EXPECT_EQ( strings_view.size(), count);
    EXPECT_EQ( strings_view.offsets().size(), count+1 );
    EXPECT_EQ( strings_view.chars().size(), memsize );

    // check string data
    auto strings_data = cudf::strings::create_offsets(strings_view);
    thrust::host_vector<char> h_chars_data(strings_data.first);
    thrust::host_vector<cudf::size_type> h_offsets_data(strings_data.second);
    EXPECT_EQ( memcmp(h_buffer.data(), h_chars_data.data(), h_buffer.size()), 0 );
    EXPECT_EQ( memcmp(h_offsets.data(), h_offsets_data.data(), h_offsets.size()*sizeof(cudf::size_type)), 0);
}
