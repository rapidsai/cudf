/*
 * Copyright (c) 2019, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <tests/strings/utilities.h>
#include <cudf/column/column_factories.hpp>
#include <cudf/copying.hpp>
#include <cudf/scalar/scalar.hpp>
#include <cudf/scalar/scalar_factories.hpp>
#include <cudf/strings/strings_column_view.hpp>
#include <cudf/types.hpp>
#include <tests/utilities/base_fixture.hpp>
#include <tests/utilities/column_utilities.hpp>
#include <tests/utilities/column_wrapper.hpp>

#include <cstring>
#include <vector>

struct StringsFactoriesTest : public cudf::test::BaseFixture {
};

TEST_F(StringsFactoriesTest, CreateColumnFromPair)
{
  std::vector<const char*> h_test_strings{"the quick brown fox jumps over the lazy dog",
                                          "the fat cat lays next to the other accénted cat",
                                          "a slow moving turtlé cannot catch the bird",
                                          "which can be composéd together to form a more complete",
                                          "thé result does not include the value in the sum in",
                                          "",
                                          nullptr,
                                          "absent stop words"};

  cudf::size_type memsize = 0;
  for (auto itr = h_test_strings.begin(); itr != h_test_strings.end(); ++itr)
    memsize += *itr ? (cudf::size_type)strlen(*itr) : 0;
  cudf::size_type count = (cudf::size_type)h_test_strings.size();
  thrust::host_vector<char> h_buffer(memsize);
  thrust::device_vector<char> d_buffer(memsize);
  thrust::host_vector<thrust::pair<const char*, cudf::size_type>> strings(count);
  thrust::host_vector<cudf::size_type> h_offsets(count + 1);
  cudf::size_type offset = 0;
  cudf::size_type nulls  = 0;
  h_offsets[0]           = 0;
  for (cudf::size_type idx = 0; idx < count; ++idx) {
    const char* str = h_test_strings[idx];
    if (!str) {
      strings[idx] = thrust::pair<const char*, cudf::size_type>{nullptr, 0};
      nulls++;
    } else {
      cudf::size_type length = (cudf::size_type)strlen(str);
      memcpy(h_buffer.data() + offset, str, length);
      strings[idx] =
        thrust::pair<const char*, cudf::size_type>{d_buffer.data().get() + offset, length};
      offset += length;
    }
    h_offsets[idx + 1] = offset;
  }
  rmm::device_vector<thrust::pair<const char*, cudf::size_type>> d_strings(strings);
  CUDA_TRY(hipMemcpy(d_buffer.data().get(), h_buffer.data(), memsize, hipMemcpyHostToDevice));
  auto column = cudf::make_strings_column(d_strings);
  EXPECT_EQ(column->type(), cudf::data_type{cudf::type_id::STRING});
  EXPECT_EQ(column->null_count(), nulls);
  if (nulls) {
    EXPECT_TRUE(column->nullable());
    EXPECT_TRUE(column->has_nulls());
  }
  EXPECT_EQ(2, column->num_children());

  cudf::strings_column_view strings_view(column->view());
  EXPECT_EQ(strings_view.size(), count);
  EXPECT_EQ(strings_view.offsets().size(), count + 1);
  EXPECT_EQ(strings_view.chars().size(), memsize);

  // check string data
  auto strings_data = cudf::strings::create_offsets(strings_view);
  thrust::host_vector<char> h_chars_data(strings_data.first);
  thrust::host_vector<cudf::size_type> h_offsets_data(strings_data.second);
  EXPECT_EQ(memcmp(h_buffer.data(), h_chars_data.data(), h_buffer.size()), 0);
  EXPECT_EQ(
    memcmp(h_offsets.data(), h_offsets_data.data(), h_offsets.size() * sizeof(cudf::size_type)), 0);
}

TEST_F(StringsFactoriesTest, CreateColumnFromOffsets)
{
  std::vector<const char*> h_test_strings{"the quick brown fox jumps over the lazy dog",
                                          "the fat cat lays next to the other accénted cat",
                                          "a slow moving turtlé cannot catch the bird",
                                          "which can be composéd together to form a more complete",
                                          "thé result does not include the value in the sum in",
                                          "",
                                          nullptr,
                                          "absent stop words"};

  cudf::size_type memsize = 0;
  for (auto itr = h_test_strings.begin(); itr != h_test_strings.end(); ++itr)
    memsize += *itr ? (cudf::size_type)strlen(*itr) : 0;
  cudf::size_type count = (cudf::size_type)h_test_strings.size();
  std::vector<char> h_buffer(memsize);
  std::vector<cudf::size_type> h_offsets(count + 1);
  cudf::size_type offset         = 0;
  h_offsets[0]                   = offset;
  cudf::bitmask_type h_null_mask = 0;
  cudf::size_type null_count     = 0;
  for (cudf::size_type idx = 0; idx < count; ++idx) {
    h_null_mask     = (h_null_mask << 1);
    const char* str = h_test_strings[idx];
    if (str) {
      cudf::size_type length = (cudf::size_type)strlen(str);
      memcpy(h_buffer.data() + offset, str, length);
      offset += length;
      h_null_mask |= 1;
    } else
      null_count++;
    h_offsets[idx + 1] = offset;
  }
  std::vector<cudf::bitmask_type> h_nulls{h_null_mask};
  rmm::device_vector<char> d_buffer(h_buffer);
  rmm::device_vector<cudf::size_type> d_offsets(h_offsets);
  rmm::device_vector<cudf::bitmask_type> d_nulls(h_nulls);
  auto column = cudf::make_strings_column(d_buffer, d_offsets, d_nulls, null_count);
  EXPECT_EQ(column->type(), cudf::data_type{cudf::type_id::STRING});
  EXPECT_EQ(column->null_count(), null_count);
  EXPECT_EQ(2, column->num_children());

  cudf::strings_column_view strings_view(column->view());
  EXPECT_EQ(strings_view.size(), count);
  EXPECT_EQ(strings_view.offsets().size(), count + 1);
  EXPECT_EQ(strings_view.chars().size(), memsize);

  // check string data
  auto strings_data = cudf::strings::create_offsets(strings_view);
  thrust::host_vector<char> h_chars_data(strings_data.first);
  thrust::host_vector<cudf::size_type> h_offsets_data(strings_data.second);
  EXPECT_EQ(memcmp(h_buffer.data(), h_chars_data.data(), h_buffer.size()), 0);
  EXPECT_EQ(
    memcmp(h_offsets.data(), h_offsets_data.data(), h_offsets.size() * sizeof(cudf::size_type)), 0);

  // check host version of the factory too
  auto column2 = cudf::make_strings_column(h_buffer, h_offsets, h_nulls, null_count);
  CUDF_TEST_EXPECT_COLUMNS_EQUAL(column->view(), column2->view());
}

TEST_F(StringsFactoriesTest, CreateScalar)
{
  std::string value = "test string";
  auto s            = cudf::make_string_scalar(value);
  auto string_s     = static_cast<cudf::string_scalar*>(s.get());

  EXPECT_EQ(string_s->to_string(), value);
  EXPECT_TRUE(string_s->is_valid());
  EXPECT_TRUE(s->is_valid());
}

TEST_F(StringsFactoriesTest, EmptyStringsColumn)
{
  rmm::device_vector<char> d_chars;
  rmm::device_vector<cudf::size_type> d_offsets(1, 0);
  rmm::device_vector<cudf::bitmask_type> d_nulls;

  auto results = cudf::make_strings_column(d_chars, d_offsets, d_nulls, 0);
  cudf::test::expect_strings_empty(results->view());

  rmm::device_vector<thrust::pair<const char*, cudf::size_type>> d_strings;
  results = cudf::make_strings_column(d_strings);
  cudf::test::expect_strings_empty(results->view());
}

TEST_F(StringsFactoriesTest, CreateOffsets)
{
  std::vector<std::string> strings      = {"this", "is", "a", "column", "of", "strings"};
  cudf::test::strings_column_wrapper sw = {strings.begin(), strings.end()};
  cudf::column_view col(sw);
  std::vector<cudf::size_type> indices{0, 2, 3, 6};
  auto result = cudf::slice(col, indices);

  std::vector<std::vector<std::string>> expecteds{
    std::vector<std::string>{"this", "is"},              // [0,2)
    std::vector<std::string>{"column", "of", "strings"}  // [3,6)
  };
  for (size_t idx = 0; idx < result.size(); idx++) {
    auto strings_data = cudf::strings::create_offsets(cudf::strings_column_view(result[idx]));
    thrust::host_vector<char> h_chars(strings_data.first);
    thrust::host_vector<cudf::size_type> h_offsets(strings_data.second);
    auto expected_strings = expecteds[idx];
    for (size_t jdx = 0; jdx < h_offsets.size() - 1; ++jdx) {
      auto offset = h_offsets[jdx];
      auto length = h_offsets[jdx + 1] - offset;
      std::string str(h_chars.data() + offset, length);
      EXPECT_EQ(str, expected_strings[jdx]);
    }
  }
}
