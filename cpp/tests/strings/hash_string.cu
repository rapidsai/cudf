/*
 * Copyright (c) 2019, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <cudf/column/column_device_view.cuh>
#include <cudf/column/column_factories.hpp>
#include <cudf/detail/utilities/hash_functions.cuh>
#include <cudf/strings/strings_column_view.hpp>
#include <cudf_test/base_fixture.hpp>
#include <cudf_test/column_wrapper.hpp>
#include "./utilities.h"

#include <thrust/execution_policy.h>
#include <thrust/transform.h>
#include <vector>

struct StringsHashTest : public cudf::test::BaseFixture {
};

struct hash_string_fn {
  cudf::column_device_view d_strings;
  uint32_t __device__ operator()(uint32_t idx)
  {
    if (d_strings.is_null(idx)) return 0;
    auto item = d_strings.element<cudf::string_view>(idx);
    return MurmurHash3_32<cudf::string_view>{}(item);
  }
};

TEST_F(StringsHashTest, HashTest)
{
  std::vector<const char*> h_strings{"abcdefghijklmnopqrstuvwxyz",
                                     "abcdefghijklmnopqrstuvwxyz",
                                     "ABCDEFGHIJKLMNOPQRSTUVWXYZ",
                                     "0123456789",
                                     "4",
                                     "",
                                     nullptr,
                                     "last one"};
  cudf::test::strings_column_wrapper strings(
    h_strings.begin(),
    h_strings.end(),
    thrust::make_transform_iterator(h_strings.begin(), [](auto str) { return str != nullptr; }));

  auto strings_view   = cudf::strings_column_view(strings);
  auto strings_column = cudf::column_device_view::create(strings_view.parent());
  auto d_view         = *strings_column;

  thrust::device_vector<uint32_t> d_values(strings_view.size());
  thrust::transform(thrust::device,
                    thrust::make_counting_iterator<uint32_t>(0),
                    thrust::make_counting_iterator<uint32_t>(strings_view.size()),
                    d_values.begin(),
                    hash_string_fn{d_view});

  uint32_t h_expected[] = {
    2739798893, 2739798893, 3506676360, 1891213601, 3778137224, 0, 0, 1551088011};
  thrust::host_vector<uint32_t> h_values(d_values);
  for (uint32_t idx = 0; idx < h_values.size(); ++idx) EXPECT_EQ(h_values[idx], h_expected[idx]);
}
