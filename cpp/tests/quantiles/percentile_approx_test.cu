#include <arrow/util/tdigest.h>

#include <cudf/detail/tdigest/tdigest.hpp>
#include <cudf/detail/valid_if.cuh>
#include <cudf/groupby.hpp>
#include <cudf/quantiles.hpp>
#include <cudf/transform.hpp>
#include <cudf/unary.hpp>

#include <cudf_test/base_fixture.hpp>
#include <cudf_test/column_wrapper.hpp>
#include <cudf_test/type_list_utilities.hpp>
#include <cudf_test/type_lists.hpp>

#include <rmm/exec_policy.hpp>

#include <tests/groupby/groupby_test_util.hpp>

using namespace cudf;

struct tdigest_gen {
  template <
    typename T,
    typename std::enable_if_t<cudf::is_numeric<T>() || cudf::is_fixed_point<T>()>* = nullptr>
  std::unique_ptr<column> operator()(column_view const& keys, column_view const& values, int delta)
  {
    cudf::table_view t({keys});
    cudf::groupby::groupby gb(t);
    std::vector<cudf::groupby::aggregation_request> requests;
    std::vector<std::unique_ptr<cudf::groupby_aggregation>> aggregations;
    aggregations.push_back(cudf::make_tdigest_aggregation<cudf::groupby_aggregation>(delta));
    requests.push_back({values, std::move(aggregations)});
    auto result = gb.aggregate(requests);
    return std::move(result.second[0].results[0]);
  }

  template <
    typename T,
    typename std::enable_if_t<!cudf::is_numeric<T>() && !cudf::is_fixed_point<T>()>* = nullptr>
  std::unique_ptr<column> operator()(column_view const& keys, column_view const& values, int delta)
  {
    CUDF_FAIL("Invalid tdigest test type");
  }
};

std::unique_ptr<column> arrow_percentile_approx(column_view const& _values,
                                                int delta,
                                                std::vector<double> const& percentages)
{
  // sort the incoming values using the same settings that groupby does.
  // this is a little weak because null_order::AFTER is hardcoded internally to groupby.
  table_view t({_values});
  auto sorted_t      = cudf::sort(t, {}, {null_order::AFTER});
  auto sorted_values = sorted_t->get_column(0).view();

  std::vector<double> h_values(sorted_values.size());
  hipMemcpy(h_values.data(),
             sorted_values.data<double>(),
             sizeof(double) * sorted_values.size(),
             hipMemcpyDeviceToHost);
  std::vector<char> h_validity(sorted_values.size());
  if (sorted_values.null_mask() != nullptr) {
    auto validity = cudf::mask_to_bools(sorted_values.null_mask(), 0, sorted_values.size());
    hipMemcpy(h_validity.data(),
               (validity->view().data<char>()),
               sizeof(char) * sorted_values.size(),
               hipMemcpyDeviceToHost);
  }

  // generate the tdigest
  arrow::internal::TDigest atd(delta, sorted_values.size() * 2);
  for (size_t idx = 0; idx < h_values.size(); idx++) {
    if (sorted_values.null_mask() == nullptr || h_validity[idx]) { atd.Add(h_values[idx]); }
  }

  // generate the percentiles and stuff them into a list column
  std::vector<double> h_result;
  h_result.reserve(percentages.size());
  std::transform(
    percentages.begin(), percentages.end(), std::back_inserter(h_result), [&atd](double p) {
      return atd.Quantile(p);
    });
  cudf::test::fixed_width_column_wrapper<double> result(h_result.begin(), h_result.end());
  cudf::test::fixed_width_column_wrapper<size_type> offsets{
    0, static_cast<size_type>(percentages.size())};
  return cudf::make_lists_column(1, offsets.release(), result.release(), 0, {});
}

struct percentile_approx_dispatch {
  template <
    typename T,
    typename std::enable_if_t<cudf::is_numeric<T>() || cudf::is_fixed_point<T>()>* = nullptr>
  std::unique_ptr<column> operator()(column_view const& keys,
                                     column_view const& values,
                                     int delta,
                                     std::vector<double> const& percentages,
                                     size_type ulps)
  {
    // arrow implementation.
    auto expected = [&]() {
      // we're explicitly casting back to doubles here but this is ok because that is
      // exactly what happens inside of the cudf implementation as values are processed as well. so
      // this should not affect results.
      auto as_doubles = cudf::cast(values, data_type{type_id::FLOAT64});
      return arrow_percentile_approx(*as_doubles, delta, percentages);
    }();

    // gpu
    cudf::table_view t({keys});
    cudf::groupby::groupby gb(t);
    std::vector<cudf::groupby::aggregation_request> requests;
    std::vector<std::unique_ptr<cudf::groupby_aggregation>> aggregations;
    aggregations.push_back(cudf::make_tdigest_aggregation<cudf::groupby_aggregation>(delta));
    requests.push_back({values, std::move(aggregations)});
    auto gb_result = gb.aggregate(requests);

    cudf::test::fixed_width_column_wrapper<double> g_percentages(percentages.begin(),
                                                                 percentages.end());
    structs_column_view scv(*(gb_result.second[0].results[0]));
    auto result = cudf::percentile_approx(scv, g_percentages);

    cudf::test::expect_columns_equivalent(
      *expected, *result, cudf::test::debug_output_level::FIRST_ERROR, ulps);

    return result;
  }

  template <
    typename T,
    typename std::enable_if_t<!cudf::is_numeric<T>() && !cudf::is_fixed_point<T>()>* = nullptr>
  std::unique_ptr<column> operator()(column_view const& keys,
                                     column_view const& values,
                                     int delta,
                                     std::vector<double> const& percentages,
                                     size_type ulps)
  {
    CUDF_FAIL("Invalid input type for percentile_approx test");
  }
};

void percentile_approx_test(column_view const& _keys,
                            column_view const& _values,
                            int delta,
                            std::vector<double> const& percentages,
                            size_type ulps)
{
  // first pass:  validate the actual percentages we get per group.

  // produce the groups
  cudf::table_view k({_keys});
  cudf::groupby::groupby pass1_gb(k);
  cudf::table_view v({_values});
  auto groups = pass1_gb.get_groups(v);
  // slice it all up so we have keys/columns for everything.
  std::vector<column_view> keys;
  std::vector<column_view> values;
  for (size_t idx = 0; idx < groups.offsets.size() - 1; idx++) {
    auto k =
      cudf::slice(groups.keys->get_column(0), {groups.offsets[idx], groups.offsets[idx + 1]});
    keys.push_back(k[0]);

    auto v =
      cudf::slice(groups.values->get_column(0), {groups.offsets[idx], groups.offsets[idx + 1]});
    values.push_back(v[0]);
  }

  std::vector<std::unique_ptr<column>> parts;
  for (size_t idx = 0; idx < values.size(); idx++) {
    // do any casting of the input
    parts.push_back(cudf::type_dispatcher(values[idx].type(),
                                          percentile_approx_dispatch{},
                                          keys[idx],
                                          values[idx],
                                          delta,
                                          percentages,
                                          ulps));
  }
  std::vector<column_view> part_views;
  std::transform(parts.begin(),
                 parts.end(),
                 std::back_inserter(part_views),
                 [](std::unique_ptr<column> const& c) { return c->view(); });
  auto expected = cudf::concatenate(part_views);

  // second pass. run the percentile_approx with all the keys in one pass and make sure we get the
  // same results as the concatenated by-key results above

  cudf::groupby::groupby gb(k);
  std::vector<cudf::groupby::aggregation_request> requests;
  std::vector<std::unique_ptr<cudf::groupby_aggregation>> aggregations;
  aggregations.push_back(cudf::make_tdigest_aggregation<cudf::groupby_aggregation>(delta));
  requests.push_back({_values, std::move(aggregations)});
  auto gb_result = gb.aggregate(requests);

  cudf::test::fixed_width_column_wrapper<double> g_percentages(percentages.begin(),
                                                               percentages.end());
  structs_column_view scv(*(gb_result.second[0].results[0]));
  auto result = cudf::percentile_approx(scv, g_percentages);

  CUDF_TEST_EXPECT_COLUMNS_EQUIVALENT(*expected, *result);
}

void simple_test(data_type input_type, std::vector<std::pair<int, int>> params)
{
  auto values = cudf::test::generate_standardized_percentile_distribution(input_type);
  // all in the same group
  auto keys = cudf::make_fixed_width_column(
    data_type{type_id::INT32}, values->size(), mask_state::UNALLOCATED);
  thrust::fill(rmm::exec_policy(rmm::cuda_stream_default),
               keys->mutable_view().template begin<int>(),
               keys->mutable_view().template end<int>(),
               0);

  std::for_each(params.begin(), params.end(), [&](std::pair<int, int> const& params) {
    percentile_approx_test(
      *keys, *values, params.first, {0.0, 0.05, 0.25, 0.5, 0.75, 0.95, 1.0}, params.second);
  });
}

struct group_index {
  __device__ int operator()(int i) { return i / 150000; }
};

void grouped_test(data_type input_type, std::vector<std::pair<int, int>> params)
{
  auto values = cudf::test::generate_standardized_percentile_distribution(input_type);
  // all in the same group
  auto keys = cudf::make_fixed_width_column(
    data_type{type_id::INT32}, values->size(), mask_state::UNALLOCATED);
  auto i = thrust::make_counting_iterator(0);
  thrust::transform(rmm::exec_policy(rmm::cuda_stream_default),
                    i,
                    i + values->size(),
                    keys->mutable_view().template begin<int>(),
                    group_index{});

  std::for_each(params.begin(), params.end(), [&](std::pair<int, int> const& params) {
    percentile_approx_test(
      *keys, *values, params.first, {0.0, 0.05, 0.25, 0.5, 0.75, 0.95, 1.0}, params.second);
  });
}

std::pair<rmm::device_buffer, size_type> make_null_mask(column_view const& col)
{
  return cudf::detail::valid_if(thrust::make_counting_iterator<size_type>(0),
                                thrust::make_counting_iterator<size_type>(col.size()),
                                [] __device__(size_type i) { return i % 2 == 0; });
}

void simple_with_nulls_test(data_type input_type, std::vector<std::pair<int, int>> params)
{
  auto values = cudf::test::generate_standardized_percentile_distribution(input_type);
  // all in the same group
  auto keys = cudf::make_fixed_width_column(
    data_type{type_id::INT32}, values->size(), mask_state::UNALLOCATED);
  thrust::fill(rmm::exec_policy(rmm::cuda_stream_default),
               keys->mutable_view().template begin<int>(),
               keys->mutable_view().template end<int>(),
               0);

  // add a null mask
  auto mask = make_null_mask(*values);
  values->set_null_mask(mask.first, mask.second);

  std::for_each(params.begin(), params.end(), [&](std::pair<int, int> const& params) {
    percentile_approx_test(
      *keys, *values, params.first, {0.0, 0.05, 0.25, 0.5, 0.75, 0.95, 1.0}, params.second);
  });
}

void grouped_with_nulls_test(data_type input_type, std::vector<std::pair<int, int>> params)
{
  auto values = cudf::test::generate_standardized_percentile_distribution(input_type);
  // all in the same group
  auto keys = cudf::make_fixed_width_column(
    data_type{type_id::INT32}, values->size(), mask_state::UNALLOCATED);
  auto i = thrust::make_counting_iterator(0);
  thrust::transform(rmm::exec_policy(rmm::cuda_stream_default),
                    i,
                    i + values->size(),
                    keys->mutable_view().template begin<int>(),
                    group_index{});

  // add a null mask
  auto mask = make_null_mask(*values);
  values->set_null_mask(mask.first, mask.second);

  std::for_each(params.begin(), params.end(), [&](std::pair<int, int> const& params) {
    percentile_approx_test(
      *keys, *values, params.first, {0.0, 0.05, 0.25, 0.5, 0.75, 0.95, 1.0}, params.second);
  });
}

template <typename T>
data_type get_appropriate_type()
{
  if constexpr (cudf::is_fixed_point<T>()) { return data_type{cudf::type_to_id<T>(), -7}; }
  return data_type{cudf::type_to_id<T>()};
}

using PercentileApproxTypes =
  cudf::test::Concat<cudf::test::NumericTypes, cudf::test::FixedPointTypes>;

template <typename T>
struct PercentileApproxInputTypesTest : public cudf::test::BaseFixture {
};
TYPED_TEST_CASE(PercentileApproxInputTypesTest, PercentileApproxTypes);

TYPED_TEST(PercentileApproxInputTypesTest, Simple)
{
  using T               = TypeParam;
  auto const input_type = get_appropriate_type<T>();

  simple_test(input_type,
              {{1000, cudf::test::default_ulp},
               {100, cudf::test::default_ulp * 4},
               {10, cudf::test::default_ulp * 11}});
}

TYPED_TEST(PercentileApproxInputTypesTest, Grouped)
{
  using T               = TypeParam;
  auto const input_type = get_appropriate_type<T>();

  grouped_test(input_type,
               {{1000, cudf::test::default_ulp},
                {100, cudf::test::default_ulp * 2},
                {10, cudf::test::default_ulp * 10}});
}

TYPED_TEST(PercentileApproxInputTypesTest, SimpleWithNulls)
{
  using T               = TypeParam;
  auto const input_type = get_appropriate_type<T>();

  simple_with_nulls_test(input_type,
                         {{1000, cudf::test::default_ulp},
                          {100, cudf::test::default_ulp * 2},
                          {10, cudf::test::default_ulp * 11}});
}

TYPED_TEST(PercentileApproxInputTypesTest, GroupedWithNulls)
{
  using T               = TypeParam;
  auto const input_type = get_appropriate_type<T>();

  grouped_with_nulls_test(input_type,
                          {{1000, cudf::test::default_ulp},
                           {100, cudf::test::default_ulp * 2},
                           {10, cudf::test::default_ulp * 6}});
}

struct PercentileApproxTest : public cudf::test::BaseFixture {
};

TEST_F(PercentileApproxTest, EmptyInput)
{
  auto empty_ = cudf::detail::tdigest::make_empty_tdigest_column();
  cudf::test::fixed_width_column_wrapper<double> percentiles{0.0, 0.25, 0.3};

  std::vector<column_view> input;
  input.push_back(*empty_);
  input.push_back(*empty_);
  input.push_back(*empty_);
  auto empty = cudf::concatenate(input);

  structs_column_view scv(*empty);
  auto result = cudf::percentile_approx(scv, percentiles);

  cudf::test::fixed_width_column_wrapper<offset_type> offsets{0, 0, 0, 0};
  std::vector<bool> nulls{0, 0, 0};
  auto expected =
    cudf::make_lists_column(3,
                            offsets.release(),
                            cudf::make_empty_column(data_type{type_id::FLOAT64}),
                            3,
                            cudf::test::detail::make_null_mask(nulls.begin(), nulls.end()));

  CUDF_TEST_EXPECT_COLUMNS_EQUAL(*result, *expected);
}

TEST_F(PercentileApproxTest, EmptyPercentiles)
{
  auto const delta = 1000;

  cudf::test::fixed_width_column_wrapper<double> values{0, 1, 2, 3, 4, 5};
  cudf::test::fixed_width_column_wrapper<int> keys{0, 0, 0, 1, 1, 1};
  cudf::table_view t({keys});
  cudf::groupby::groupby gb(t);
  std::vector<cudf::groupby::aggregation_request> requests;
  std::vector<std::unique_ptr<cudf::groupby_aggregation>> aggregations;
  aggregations.push_back(cudf::make_tdigest_aggregation<cudf::groupby_aggregation>(delta));
  requests.push_back({values, std::move(aggregations)});
  auto tdigest_column = gb.aggregate(requests);

  cudf::test::fixed_width_column_wrapper<double> percentiles{};

  structs_column_view scv(*tdigest_column.second[0].results[0]);
  auto result = cudf::percentile_approx(scv, percentiles);

  cudf::test::fixed_width_column_wrapper<offset_type> offsets{0, 0, 0};
  auto expected = cudf::make_lists_column(2,
                                          offsets.release(),
                                          cudf::make_empty_column(data_type{type_id::FLOAT64}),
                                          2,
                                          cudf::detail::create_null_mask(2, mask_state::ALL_NULL));

  CUDF_TEST_EXPECT_COLUMNS_EQUAL(*result, *expected);
}

TEST_F(PercentileApproxTest, NullPercentiles)
{
  auto const delta = 1000;

  cudf::test::fixed_width_column_wrapper<double> values{1, 1, 2, 3, 4, 5, 6, 7, 8};
  cudf::test::fixed_width_column_wrapper<int> keys{0, 0, 0, 0, 0, 1, 1, 1, 1};
  cudf::table_view t({keys});
  cudf::groupby::groupby gb(t);
  std::vector<cudf::groupby::aggregation_request> requests;
  std::vector<std::unique_ptr<cudf::groupby_aggregation>> aggregations;
  aggregations.push_back(cudf::make_tdigest_aggregation<cudf::groupby_aggregation>(delta));
  requests.push_back({values, std::move(aggregations)});
  auto tdigest_column = gb.aggregate(requests);

  structs_column_view scv(*tdigest_column.second[0].results[0]);

  cudf::test::fixed_width_column_wrapper<double> npercentiles{{0.5, 0.5, 1.0, 1.0}, {0, 0, 1, 1}};
  auto result = cudf::percentile_approx(scv, npercentiles);

  std::vector<bool> valids{0, 0, 1, 1};
  cudf::test::lists_column_wrapper<double> expected{{{99, 99, 4, 4}, valids.begin()},
                                                    {{99, 99, 8, 8}, valids.begin()}};

  CUDF_TEST_EXPECT_COLUMNS_EQUAL(*result, expected);
}
