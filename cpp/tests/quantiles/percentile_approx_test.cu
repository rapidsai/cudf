/*
 * Copyright (c) 2022, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */
#include <arrow/util/tdigest.h>

#include <cudf/detail/tdigest/tdigest.hpp>
#include <cudf/detail/valid_if.cuh>
#include <cudf/groupby.hpp>
#include <cudf/quantiles.hpp>
#include <cudf/reduction.hpp>
#include <cudf/tdigest/tdigest_column_view.cuh>
#include <cudf/transform.hpp>
#include <cudf/unary.hpp>
#include <cudf/utilities/default_stream.hpp>

#include <cudf_test/base_fixture.hpp>
#include <cudf_test/column_wrapper.hpp>
#include <cudf_test/type_list_utilities.hpp>
#include <cudf_test/type_lists.hpp>

#include <rmm/exec_policy.hpp>

#include <tests/groupby/groupby_test_util.hpp>

#include <thrust/fill.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/transform.h>

using namespace cudf;
using namespace cudf::tdigest;

std::unique_ptr<column> arrow_percentile_approx(column_view const& _values,
                                                int delta,
                                                std::vector<double> const& percentages)
{
  // sort the incoming values using the same settings that groupby does.
  // this is a little weak because null_order::AFTER is hardcoded internally to groupby.
  table_view t({_values});
  auto sorted_t      = cudf::sort(t, {}, {null_order::AFTER});
  auto sorted_values = sorted_t->get_column(0).view();

  std::vector<double> h_values(sorted_values.size());
  hipMemcpy(h_values.data(),
             sorted_values.data<double>(),
             sizeof(double) * sorted_values.size(),
             hipMemcpyDeviceToHost);
  std::vector<char> h_validity(sorted_values.size());
  if (sorted_values.null_mask() != nullptr) {
    auto validity = cudf::mask_to_bools(sorted_values.null_mask(), 0, sorted_values.size());
    hipMemcpy(h_validity.data(),
               (validity->view().data<char>()),
               sizeof(char) * sorted_values.size(),
               hipMemcpyDeviceToHost);
  }

  // generate the tdigest
  arrow::internal::TDigest atd(delta, sorted_values.size() * 2);
  for (size_t idx = 0; idx < h_values.size(); idx++) {
    if (sorted_values.null_mask() == nullptr || h_validity[idx]) { atd.Add(h_values[idx]); }
  }

  // generate the percentiles and stuff them into a list column
  std::vector<double> h_result;
  h_result.reserve(percentages.size());
  std::transform(
    percentages.begin(), percentages.end(), std::back_inserter(h_result), [&atd](double p) {
      return atd.Quantile(p);
    });
  cudf::test::fixed_width_column_wrapper<double> result(h_result.begin(), h_result.end());
  cudf::test::fixed_width_column_wrapper<size_type> offsets{
    0, static_cast<size_type>(percentages.size())};
  return cudf::make_lists_column(1, offsets.release(), result.release(), 0, {});
}

struct percentile_approx_dispatch {
  template <
    typename T,
    typename Func,
    typename std::enable_if_t<cudf::is_numeric<T>() || cudf::is_fixed_point<T>()>* = nullptr>
  std::unique_ptr<column> operator()(Func op,
                                     column_view const& values,
                                     int delta,
                                     std::vector<double> const& percentages,
                                     size_type ulps)
  {
    // arrow implementation.
    auto expected = [&]() {
      // we're explicitly casting back to doubles here but this is ok because that is
      // exactly what happens inside of the cudf implementation as values are processed as well. so
      // this should not affect results.
      auto as_doubles = cudf::cast(values, data_type{type_id::FLOAT64});
      return arrow_percentile_approx(*as_doubles, delta, percentages);
    }();

    // gpu implementation
    auto agg_result = op(values, delta);

    cudf::test::fixed_width_column_wrapper<double> g_percentages(percentages.begin(),
                                                                 percentages.end());
    tdigest_column_view tdv(*agg_result);
    auto result = cudf::percentile_approx(tdv, g_percentages);

    cudf::test::expect_columns_equivalent(
      *expected, *result, cudf::test::debug_output_level::FIRST_ERROR, ulps);

    return result;
  }

  template <
    typename T,
    typename Func,
    typename std::enable_if_t<!cudf::is_numeric<T>() && !cudf::is_fixed_point<T>()>* = nullptr>
  std::unique_ptr<column> operator()(Func op,
                                     column_view const& values,
                                     int delta,
                                     std::vector<double> const& percentages,
                                     size_type ulps)
  {
    CUDF_FAIL("Invalid input type for percentile_approx test");
  }
};

void percentile_approx_test(column_view const& _keys,
                            column_view const& _values,
                            int delta,
                            std::vector<double> const& percentages,
                            size_type ulps)
{
  // first pass:  validate the actual percentages we get per group.

  // produce the groups.
  cudf::table_view k({_keys});
  cudf::groupby::groupby pass1_gb(k);
  cudf::table_view v({_values});
  auto groups = pass1_gb.get_groups(v);
  // slice it all up so we have keys/columns for everything.
  std::vector<column_view> keys;
  std::vector<column_view> values;
  for (size_t idx = 0; idx < groups.offsets.size() - 1; idx++) {
    auto k =
      cudf::slice(groups.keys->get_column(0), {groups.offsets[idx], groups.offsets[idx + 1]});
    keys.push_back(k[0]);

    auto v =
      cudf::slice(groups.values->get_column(0), {groups.offsets[idx], groups.offsets[idx + 1]});
    values.push_back(v[0]);
  }

  std::vector<std::unique_ptr<column>> groupby_parts;
  std::vector<std::unique_ptr<column>> reduce_parts;
  for (size_t idx = 0; idx < values.size(); idx++) {
    // via groupby
    auto groupby = [&](column_view const& values, int delta) {
      cudf::table_view t({keys[idx]});
      cudf::groupby::groupby gb(t);
      std::vector<cudf::groupby::aggregation_request> requests;
      std::vector<std::unique_ptr<cudf::groupby_aggregation>> aggregations;
      aggregations.push_back(cudf::make_tdigest_aggregation<cudf::groupby_aggregation>(delta));
      requests.push_back({values, std::move(aggregations)});
      return std::move(gb.aggregate(requests).second[0].results[0]);
    };
    groupby_parts.push_back(cudf::type_dispatcher(values[idx].type(),
                                                  percentile_approx_dispatch{},
                                                  groupby,
                                                  values[idx],
                                                  delta,
                                                  percentages,
                                                  ulps));

    // via reduce
    auto reduce = [](column_view const& values, int delta) {
      // result is a scalar, but we want to extract out the underlying column
      auto scalar_result =
        cudf::reduce(values,
                     *cudf::make_tdigest_aggregation<cudf::reduce_aggregation>(delta),
                     data_type{type_id::STRUCT});
      auto tbl = static_cast<cudf::struct_scalar const*>(scalar_result.get())->view();
      std::vector<std::unique_ptr<cudf::column>> cols;
      std::transform(
        tbl.begin(), tbl.end(), std::back_inserter(cols), [](cudf::column_view const& col) {
          return std::make_unique<cudf::column>(col);
        });
      return cudf::make_structs_column(tbl.num_rows(), std::move(cols), 0, rmm::device_buffer());
    };
    // groupby path
    reduce_parts.push_back(cudf::type_dispatcher(values[idx].type(),
                                                 percentile_approx_dispatch{},
                                                 reduce,
                                                 values[idx],
                                                 delta,
                                                 percentages,
                                                 ulps));
  }

  // second pass. run the percentile_approx with all the keys in one pass and make sure we get the
  // same results as the concatenated by-key results.
  std::vector<column_view> part_views;
  std::transform(groupby_parts.begin(),
                 groupby_parts.end(),
                 std::back_inserter(part_views),
                 [](std::unique_ptr<column> const& c) { return c->view(); });
  auto expected = cudf::concatenate(part_views);

  cudf::groupby::groupby gb(k);
  std::vector<cudf::groupby::aggregation_request> requests;
  std::vector<std::unique_ptr<cudf::groupby_aggregation>> aggregations;
  aggregations.push_back(cudf::make_tdigest_aggregation<cudf::groupby_aggregation>(delta));
  requests.push_back({_values, std::move(aggregations)});
  auto gb_result = gb.aggregate(requests);

  cudf::test::fixed_width_column_wrapper<double> g_percentages(percentages.begin(),
                                                               percentages.end());
  tdigest_column_view tdv(*(gb_result.second[0].results[0]));
  auto result = cudf::percentile_approx(tdv, g_percentages);

  CUDF_TEST_EXPECT_COLUMNS_EQUIVALENT(*expected, *result);
}

void simple_test(data_type input_type, std::vector<std::pair<int, int>> params)
{
  auto values = cudf::test::generate_standardized_percentile_distribution(input_type);
  // all in the same group
  auto keys = cudf::make_fixed_width_column(
    data_type{type_id::INT32}, values->size(), mask_state::UNALLOCATED);
  thrust::fill(rmm::exec_policy(cudf::get_default_stream()),
               keys->mutable_view().template begin<int>(),
               keys->mutable_view().template end<int>(),
               0);

  // runs both groupby and reduce paths
  std::for_each(params.begin(), params.end(), [&](std::pair<int, int> const& params) {
    percentile_approx_test(
      *keys, *values, params.first, {0.0, 0.05, 0.25, 0.5, 0.75, 0.95, 1.0}, params.second);
  });
}

struct group_index {
  __device__ int operator()(int i) { return i / 150000; }
};

void grouped_test(data_type input_type, std::vector<std::pair<int, int>> params)
{
  auto values = cudf::test::generate_standardized_percentile_distribution(input_type);
  // all in the same group
  auto keys = cudf::make_fixed_width_column(
    data_type{type_id::INT32}, values->size(), mask_state::UNALLOCATED);
  auto i = thrust::make_counting_iterator(0);
  thrust::transform(rmm::exec_policy(cudf::get_default_stream()),
                    i,
                    i + values->size(),
                    keys->mutable_view().template begin<int>(),
                    group_index{});

  std::for_each(params.begin(), params.end(), [&](std::pair<int, int> const& params) {
    percentile_approx_test(
      *keys, *values, params.first, {0.0, 0.05, 0.25, 0.5, 0.75, 0.95, 1.0}, params.second);
  });
}

std::pair<rmm::device_buffer, size_type> make_null_mask(column_view const& col)
{
  return cudf::detail::valid_if(thrust::make_counting_iterator<size_type>(0),
                                thrust::make_counting_iterator<size_type>(col.size()),
                                [] __device__(size_type i) { return i % 2 == 0; });
}

void simple_with_nulls_test(data_type input_type, std::vector<std::pair<int, int>> params)
{
  auto values = cudf::test::generate_standardized_percentile_distribution(input_type);
  // all in the same group
  auto keys = cudf::make_fixed_width_column(
    data_type{type_id::INT32}, values->size(), mask_state::UNALLOCATED);
  thrust::fill(rmm::exec_policy(cudf::get_default_stream()),
               keys->mutable_view().template begin<int>(),
               keys->mutable_view().template end<int>(),
               0);

  // add a null mask
  auto mask = make_null_mask(*values);
  values->set_null_mask(mask.first, mask.second);

  std::for_each(params.begin(), params.end(), [&](std::pair<int, int> const& params) {
    percentile_approx_test(
      *keys, *values, params.first, {0.0, 0.05, 0.25, 0.5, 0.75, 0.95, 1.0}, params.second);
  });
}

void grouped_with_nulls_test(data_type input_type, std::vector<std::pair<int, int>> params)
{
  auto values = cudf::test::generate_standardized_percentile_distribution(input_type);
  // all in the same group
  auto keys = cudf::make_fixed_width_column(
    data_type{type_id::INT32}, values->size(), mask_state::UNALLOCATED);
  auto i = thrust::make_counting_iterator(0);
  thrust::transform(rmm::exec_policy(cudf::get_default_stream()),
                    i,
                    i + values->size(),
                    keys->mutable_view().template begin<int>(),
                    group_index{});

  // add a null mask
  auto mask = make_null_mask(*values);
  values->set_null_mask(mask.first, mask.second);

  std::for_each(params.begin(), params.end(), [&](std::pair<int, int> const& params) {
    percentile_approx_test(
      *keys, *values, params.first, {0.0, 0.05, 0.25, 0.5, 0.75, 0.95, 1.0}, params.second);
  });
}

template <typename T>
data_type get_appropriate_type()
{
  if constexpr (cudf::is_fixed_point<T>()) { return data_type{cudf::type_to_id<T>(), -7}; }
  return data_type{cudf::type_to_id<T>()};
}

using PercentileApproxTypes =
  cudf::test::Concat<cudf::test::NumericTypes, cudf::test::FixedPointTypes>;

template <typename T>
struct PercentileApproxInputTypesTest : public cudf::test::BaseFixture {
};
TYPED_TEST_SUITE(PercentileApproxInputTypesTest, PercentileApproxTypes);

TYPED_TEST(PercentileApproxInputTypesTest, Simple)
{
  using T               = TypeParam;
  auto const input_type = get_appropriate_type<T>();

  simple_test(input_type,
              {{1000, cudf::test::default_ulp},
               {100, cudf::test::default_ulp * 4},
               {10, cudf::test::default_ulp * 11}});
}

TYPED_TEST(PercentileApproxInputTypesTest, Grouped)
{
  using T               = TypeParam;
  auto const input_type = get_appropriate_type<T>();

  grouped_test(input_type,
               {{1000, cudf::test::default_ulp},
                {100, cudf::test::default_ulp * 2},
                {10, cudf::test::default_ulp * 10}});
}

TYPED_TEST(PercentileApproxInputTypesTest, SimpleWithNulls)
{
  using T               = TypeParam;
  auto const input_type = get_appropriate_type<T>();

  simple_with_nulls_test(input_type,
                         {{1000, cudf::test::default_ulp},
                          {100, cudf::test::default_ulp * 2},
                          {10, cudf::test::default_ulp * 11}});
}

TYPED_TEST(PercentileApproxInputTypesTest, GroupedWithNulls)
{
  using T               = TypeParam;
  auto const input_type = get_appropriate_type<T>();

  grouped_with_nulls_test(input_type,
                          {{1000, cudf::test::default_ulp},
                           {100, cudf::test::default_ulp * 2},
                           {10, cudf::test::default_ulp * 6}});
}

struct PercentileApproxTest : public cudf::test::BaseFixture {
};

TEST_F(PercentileApproxTest, EmptyInput)
{
  auto empty_ = cudf::detail::tdigest::make_empty_tdigest_column();
  cudf::test::fixed_width_column_wrapper<double> percentiles{0.0, 0.25, 0.3};

  std::vector<column_view> input;
  input.push_back(*empty_);
  input.push_back(*empty_);
  input.push_back(*empty_);
  auto empty = cudf::concatenate(input);

  tdigest_column_view tdv(*empty);
  auto result = cudf::percentile_approx(tdv, percentiles);

  cudf::test::fixed_width_column_wrapper<offset_type> offsets{0, 0, 0, 0};
  std::vector<bool> nulls{0, 0, 0};
  auto expected =
    cudf::make_lists_column(3,
                            offsets.release(),
                            cudf::make_empty_column(type_id::FLOAT64),
                            3,
                            cudf::test::detail::make_null_mask(nulls.begin(), nulls.end()));

  CUDF_TEST_EXPECT_COLUMNS_EQUAL(*result, *expected);
}

TEST_F(PercentileApproxTest, EmptyPercentiles)
{
  auto const delta = 1000;

  cudf::test::fixed_width_column_wrapper<double> values{0, 1, 2, 3, 4, 5};
  cudf::test::fixed_width_column_wrapper<int> keys{0, 0, 0, 1, 1, 1};
  cudf::table_view t({keys});
  cudf::groupby::groupby gb(t);
  std::vector<cudf::groupby::aggregation_request> requests;
  std::vector<std::unique_ptr<cudf::groupby_aggregation>> aggregations;
  aggregations.push_back(cudf::make_tdigest_aggregation<cudf::groupby_aggregation>(delta));
  requests.push_back({values, std::move(aggregations)});
  auto tdigest_column = gb.aggregate(requests);

  cudf::test::fixed_width_column_wrapper<double> percentiles{};

  tdigest_column_view tdv(*tdigest_column.second[0].results[0]);
  auto result = cudf::percentile_approx(tdv, percentiles);

  cudf::test::fixed_width_column_wrapper<offset_type> offsets{0, 0, 0};
  auto expected = cudf::make_lists_column(2,
                                          offsets.release(),
                                          cudf::make_empty_column(type_id::FLOAT64),
                                          2,
                                          cudf::detail::create_null_mask(2, mask_state::ALL_NULL));

  CUDF_TEST_EXPECT_COLUMNS_EQUAL(*result, *expected);
}

TEST_F(PercentileApproxTest, NullPercentiles)
{
  auto const delta = 1000;

  cudf::test::fixed_width_column_wrapper<double> values{1, 1, 2, 3, 4, 5, 6, 7, 8};
  cudf::test::fixed_width_column_wrapper<int> keys{0, 0, 0, 0, 0, 1, 1, 1, 1};
  cudf::table_view t({keys});
  cudf::groupby::groupby gb(t);
  std::vector<cudf::groupby::aggregation_request> requests;
  std::vector<std::unique_ptr<cudf::groupby_aggregation>> aggregations;
  aggregations.push_back(cudf::make_tdigest_aggregation<cudf::groupby_aggregation>(delta));
  requests.push_back({values, std::move(aggregations)});
  auto tdigest_column = gb.aggregate(requests);

  tdigest_column_view tdv(*tdigest_column.second[0].results[0]);

  cudf::test::fixed_width_column_wrapper<double> npercentiles{{0.5, 0.5, 1.0, 1.0}, {0, 0, 1, 1}};
  auto result = cudf::percentile_approx(tdv, npercentiles);

  std::vector<bool> valids{0, 0, 1, 1};
  cudf::test::lists_column_wrapper<double> expected{{{99, 99, 4, 4}, valids.begin()},
                                                    {{99, 99, 8, 8}, valids.begin()}};

  CUDF_TEST_EXPECT_COLUMNS_EQUAL(*result, expected);
}
