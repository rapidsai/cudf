#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2018, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

//Quantile (percentile) testing


#include <thrust/device_vector.h>
#include <thrust/copy.h>

#include <iostream>
#include <vector>
#include <string>

#include <cassert>
#include <cmath>

#include "gtest/gtest.h"

#include <cudf.h>
#include <cudf/functions.h>
#include <utilities/cudf_utils.h>
#include <rmm/thrust_rmm_allocator.h>
#include <utilities/error_utils.h>
#include <quantiles/quantiles.h>

#include "tests/utilities/cudf_test_fixtures.h"



template<typename T, typename Allocator, template<typename, typename> class Vector>
__host__ __device__
void print_v(const Vector<T, Allocator>& v, std::ostream& os)
{
  thrust::copy(v.begin(), v.end(), std::ostream_iterator<T>(os,","));
  os<<"\n";
}


template<typename VType>
void f_quantile_tester(gdf_column* col_in, std::vector<VType>& v_out_exact, std::vector<std::vector<double>>& v_out_m, const gdf_error expected_error = GDF_SUCCESS)
{
  std::vector<std::string> methods{"lin_interp", "lower", "higher", "midpoint", "nearest"};
  size_t n_methods = methods.size();
  
  std::vector<double> qvals{0.0, 0.25, 0.33, 0.5, 1.0};
  size_t n_qs = qvals.size();
  
  assert( n_methods == methods.size() );
  gdf_context ctxt{0, static_cast<gdf_method>(0), 0, 1};
  
  for(size_t j = 0; j<n_qs; ++j)
    {
      VType res = 0;
      auto q = qvals[j];
      gdf_error ret = gdf_quantile_aprrox(col_in, q, &res, &ctxt);
      v_out_exact[j] = res;
      EXPECT_EQ( ret, expected_error) << "approx " << " returns unexpected failure\n";
      
      for(size_t i = 0;i<n_methods;++i)
        {
          double rt = 0;
          ret = gdf_quantile_exact(col_in, static_cast<gdf_quantile_method>(i), q, &rt, &ctxt);
          v_out_m[j][i] = rt;
          
          EXPECT_EQ( ret, expected_error) << "exact " << methods[i] << " returns unexpected failure\n";
        }
    }
}

struct gdf_quantile : public GdfTest {};

TEST_F(gdf_quantile, DoubleVector)
{
  using VType = double;
  std::vector<VType> v{6.8, 0.15, 3.4, 4.17, 2.13, 1.11, -1.01, 0.8, 5.7};
  rmm::device_vector<VType> d_in = v;
  rmm::device_vector<gdf_valid_type> d_valid(gdf_get_num_chars_bitmask(d_in.size()));
  
  gdf_column col_in;
  col_in.size = d_in.size();
  col_in.data = d_in.data().get();
  col_in.valid = d_valid.data().get();
  col_in.null_count = 0;
  col_in.dtype = GDF_FLOAT64;

  size_t n_qs = 5;
  size_t n_methods = 5;

  std::vector<VType> v_baseline_approx{-1.01, 0.15, 0.15, 1.11, 6.8};
  std::vector<std::vector<double>> v_baseline_exact{
    {-1.01, -1.01, 0.15, -0.43, -1.01},
      {0.3125, 0.15, 0.8, 0.475, 0.15},
        {0.7805, 0.15, 0.8, 0.475, 0.8},
          {1.62, 1.11, 2.13, 1.62, 2.13},
            {6.8, 6.8, 6.8, 6.8, 6.8}};
  
  std::vector<VType> v_out_approx(n_qs, 0);
  std::vector<std::vector<double>> v_out_exact(n_qs, std::vector<double>(n_methods,0.0));

  f_quantile_tester<VType>(&col_in, v_out_approx, v_out_exact);

  for(size_t i=0; i<n_qs;++i)
    {
      double delta = std::abs(static_cast<double>(v_baseline_approx[i] - v_out_approx[i]));
      bool flag = delta < 1.0e-8;
      EXPECT_EQ( flag, true ) << i <<"-th quantile deviates from baseline by: " << delta;
    }

  for(size_t i=0; i<n_qs;++i)
    {
      for(size_t j=0; j < n_methods; ++j)
        {
          double delta = std::abs(static_cast<double>(v_baseline_exact[i][j] - v_out_exact[i][j]));
          bool flag = delta < 1.0e-8;
          EXPECT_EQ( flag, true ) << i <<"-th quantile on " << j << "-th deviates from baseline by: " << delta;
        }
    }
}

TEST_F(gdf_quantile, IntegerVector)
{
  using VType = int32_t;
  std::vector<VType> v{7, 0, 3, 4, 2, 1, -1, 1, 6};;
  rmm::device_vector<VType> d_in = v;
  rmm::device_vector<gdf_valid_type> d_valid(gdf_get_num_chars_bitmask(d_in.size()));
  
  gdf_column col_in;
  col_in.size = d_in.size();
  col_in.data = d_in.data().get();
  col_in.valid = d_valid.data().get();
  col_in.null_count = 0;
  col_in.dtype = GDF_INT32;

  size_t n_qs = 5;
  size_t n_methods = 5;

  std::vector<VType> v_baseline_approx{-1, 0, 0, 1, 7};
  std::vector<std::vector<double>> v_baseline_exact{
    {-1, -1, 0, -0.5, -1},
      {0.25, 0, 1, 0.5, 0},
        {0.97, 0, 1, 0.5, 1},
          {1.5, 1, 2, 1.5, 2},
            {7, 7, 7, 7, 7}};
  
  std::vector<VType> v_out_approx(n_qs, 0);
  std::vector<std::vector<double>> v_out_exact(n_qs, std::vector<double>(n_methods,0.0));

  f_quantile_tester<VType>(&col_in, v_out_approx, v_out_exact);

  for(size_t i=0; i<n_qs;++i)
    {
      double delta = std::abs(static_cast<double>(v_baseline_approx[i] - v_out_approx[i]));
      bool flag = delta < 1.0e-8;
      EXPECT_EQ( flag, true ) << i <<"-th quantile deviates from baseline by: " << delta;
    }

  for(size_t i=0; i<n_qs;++i)
    {
      for(size_t j=0; j < n_methods; ++j)
        {
          double delta = std::abs(static_cast<double>(v_baseline_exact[i][j] - v_out_exact[i][j]));
          bool flag = delta < 1.0e-8;
          EXPECT_EQ( flag, true ) << i <<"-th quantile on " << j << "-th deviates from baseline by: " << delta;
        }
    }
}

TEST_F(gdf_quantile, ReportValidMaskError)
{
  using VType = int32_t;
  std::vector<VType> v{7, 0, 3, 4, 2, 1, -1, 1, 6};;
  rmm::device_vector<VType> d_in = v;
  rmm::device_vector<gdf_valid_type> d_valid(gdf_get_num_chars_bitmask(d_in.size()));
  
  gdf_column col_in;
  col_in.size = d_in.size();
  col_in.data = d_in.data().get();
  col_in.valid = d_valid.data().get();
  col_in.null_count = 1;//Should cause the quantile calls to fail
  col_in.dtype = GDF_INT32;

  size_t n_qs = 5;
  size_t n_methods = 5;

  std::vector<VType> v_baseline_approx{-1, 0, 0, 1, 7};
  std::vector<std::vector<double>> v_baseline_exact{
      {-1, -1, 0, -0.5, -1},
      {0.25, 0, 1, 0.5, 0},
      {0.97, 0, 1, 0.5, 1},
      {1.5, 1, 2, 1.5, 2},
      {7, 7, 7, 7, 7}};
  
  std::vector<VType> v_out_approx(n_qs, 0);
  std::vector<std::vector<double>> v_out_exact(n_qs, std::vector<double>(n_methods,0.0));

  f_quantile_tester<VType>(&col_in, v_out_approx, v_out_exact, GDF_VALIDITY_UNSUPPORTED);
}


int main(int argc, char **argv) {
  ::testing::InitGoogleTest(&argc, argv);
  return RUN_ALL_TESTS();
}


