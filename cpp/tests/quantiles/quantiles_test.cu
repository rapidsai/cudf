#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <cudf/scalar/scalar.hpp>
#include <cudf/table/table_view.hpp>
#include <cudf/types.hpp>
#include <cudf/quantiles.hpp>
#include <limits>
#include <memory>
#include <tests/utilities/base_fixture.hpp>
#include <tests/utilities/column_wrapper.hpp>
#include <tests/utilities/scalar_utilities.hpp>
#include <tests/utilities/type_list_utilities.hpp>
#include <tests/utilities/type_lists.hpp>
#include <type_traits>

using namespace cudf::test;

using std::vector;
using cudf::experimental::bool8;
using cudf::null_order;
using cudf::order;

namespace {

using q_res = cudf::numeric_scalar<double>;

// ----- test data -------------------------------------------------------------

namespace testdata {

    struct q_expect
    {
        q_expect(double quantile):
            quantile(quantile),
            higher(0, false), lower(0, false), linear(0, false), midpoint(0, false), nearest(0, false) { }

        q_expect(double quantile,
                 double higher, double lower, double linear, double midpoint, double nearest):
            quantile(quantile),
            higher(higher), lower(lower), linear(linear), midpoint(midpoint), nearest(nearest) { }
    
        double quantile;
        q_res higher;
        q_res lower;
        q_res linear;
        q_res midpoint;
        q_res nearest;
    };

template<typename T>
struct test_case {
    fixed_width_column_wrapper<T> column;
    vector<q_expect> expectations;
    cudf::order_info column_order;
};

// empty

template<typename T>
test_case<T>
empty() {
    return test_case<T> {
        fixed_width_column_wrapper<T> ({ }),
        {
            q_expect{ -1.0 },
            q_expect{  0.0 },
            q_expect{  0.5 },
            q_expect{  1.0 },
            q_expect{  2.0 }
        }
    };
}

// interpolate_center

template<typename T>
test_case<T>
interpolate_center() {
    auto low = std::numeric_limits<T>::lowest();
    auto max = std::numeric_limits<T>::max();
    auto mid_d = std::is_floating_point<T>::value ? 0.0 : -0.5;

    // int64_t is internally casted to a double, meaning the lerp center point
    // is float-like.
    auto lin_d = std::is_floating_point<T>::value ||
                 std::is_same<T, int64_t>::value ? 0.0 : -0.5;
    auto max_d = static_cast<double>(max);
    auto low_d = static_cast<double>(low);
    return test_case<T> {
        fixed_width_column_wrapper<T> ({ low, max }),
        {
            q_expect{ 0.50, max_d, low_d, lin_d, mid_d, low_d }
        }
    };
}

template<>
test_case<bool8>
interpolate_center() {
    auto low = std::numeric_limits<bool8>::lowest();
    auto max = std::numeric_limits<bool8>::max();
    auto mid_d = 0.5;
    auto low_d = static_cast<double>(low);
    auto max_d = static_cast<double>(max);
    return test_case<bool8> {
        fixed_width_column_wrapper<bool8> ({ low, max }),
        {
            q_expect{ 0.5, max_d, low_d, mid_d, mid_d, low_d }
        }
    };
}

// interpolate_extrema_high

template<typename T>
test_case<T>
interpolate_extrema_high() {
    T max = std::numeric_limits<T>::max();
    T low = max - 2;
    auto low_d = static_cast<double>(low);
    auto max_d = static_cast<double>(max);
    auto exact_d = static_cast<double>(max - 1);
    return test_case<T> {
        fixed_width_column_wrapper<T> ({ low, max }),
        {
            q_expect{ 0.50, max_d, low_d, exact_d, exact_d, low_d }
        }
    };
}

template<>
test_case<bool8>
interpolate_extrema_high<bool8>() {
    return interpolate_center<bool8>();
}

// interpolate_extrema_low

template<typename T>
test_case<T>
interpolate_extrema_low() {
    T lowest = std::numeric_limits<T>::lowest();
    T a = lowest;
    T b = lowest + 2;
    auto a_d = static_cast<double>(a);
    auto b_d = static_cast<double>(b);
    auto exact_d = static_cast<double>(a + 1);
    return test_case<T> {
        fixed_width_column_wrapper<T> ({ a, b }),
        {
            q_expect{ 0.50, b_d, a_d, exact_d, exact_d, a_d }
        }
    };
}

template<>
test_case<bool8>
interpolate_extrema_low<bool8>() {
    return interpolate_center<bool8>();
}

// sorted_ascending_null_before

template<typename T>
std::enable_if_t<std::is_floating_point<T>::value, test_case<T>>
sorted_ascending_null_before() {
    return test_case<T> {
        fixed_width_column_wrapper<T> ({ 1, 2, 3, 4, 5, 6, 7, 8, 9 },
                                       { 0, 0, 0, 0, 0, 1, 1, 1, 1 }),
        {
            q_expect{ 0.00, 6, 6, 6, 6, 6 },
            q_expect{ 0.75, 9, 8, 8.25, 8.5, 8 },
            q_expect{ 1.00, 9, 9, 9, 9, 9 }
        },
        { true, cudf::order::ASCENDING, cudf::null_order::BEFORE }
    };
}

template<typename T>
std::enable_if_t<std::is_integral<T>::value and not cudf::is_boolean<T>(), test_case<T>>
sorted_ascending_null_before() {
    return test_case<T> {
        fixed_width_column_wrapper<T> ({ 1, 2, 3, 4, 5, 6, 7, 8, 9 },
                                       { 0, 0, 0, 0, 0, 1, 1, 1, 1 }),
        {
            q_expect{ 0.00, 6, 6, 6, 6, 6 },
            q_expect{ 0.50, 8, 7, 7.5, 7.5, 8 },
            q_expect{ 1.00, 9, 9, 9, 9, 9 }
        },
        { true, cudf::order::ASCENDING, cudf::null_order::BEFORE }
    };
}

template<typename T>
std::enable_if_t<cudf::is_boolean<T>(), test_case<T>>
sorted_ascending_null_before() {
    return test_case<T> {
        fixed_width_column_wrapper<T> ({ 1, 0, 1, },
                                       { 0, 1, 1, }),
        {
            q_expect{ 0.00, 0, 0, 0, 0, 0 },
            q_expect{ 0.50, 1, 0, 0.5, 0.5, 0 },
            q_expect{ 1.50, 1, 1, 1, 1, 1 }
        },
        { true, cudf::order::ASCENDING, cudf::null_order::BEFORE }
    };
}

// sorted_descending_null_after

template<typename T>
std::enable_if_t<std::is_floating_point<T>::value, test_case<T>>
sorted_descending_null_after() {
    return test_case<T> {
        fixed_width_column_wrapper<T> ({ 9, 8, 7, 6, 5, 4, 3, 2, 1 },
                                       { 1, 1, 1, 1, 0, 0, 0, 0, 0 }),
        {
            q_expect{ 0.00, 6, 6, 6, 6, 6 },
            q_expect{ 0.75, 9, 8, 8.25, 8.5, 8 },
            q_expect{ 1.00, 9, 9, 9, 9, 9 }
        },
        { true, cudf::order::DESCENDING, cudf::null_order::AFTER }
    };
}

template<typename T>
std::enable_if_t<std::is_integral<T>::value and not cudf::is_boolean<T>(), test_case<T>>
sorted_descending_null_after() {
    return test_case<T> {
        fixed_width_column_wrapper<T> ({ 9, 8, 7, 6, 5, 4, 3, 2, 1 },
                                       { 1, 1, 1, 1, 0, 0, 0, 0, 0 }),
        {
            q_expect{ 0.00, 6, 6, 6, 6, 6 },
            q_expect{ 0.50, 8, 7, 7.5, 7.5, 8 },
            q_expect{ 1.00, 9, 9, 9, 9, 9 }
        },
        { true, cudf::order::DESCENDING, cudf::null_order::AFTER }
    };
}

template<typename T>
std::enable_if_t<cudf::is_boolean<T>(), test_case<T>>
sorted_descending_null_after() {
    return test_case<T> {
        fixed_width_column_wrapper<T> ({ 1, 0, 1, },
                                       { 1, 1, 0, }),
        {
            q_expect{ 0.50, 1, 0, 0.5, 0.5, 0 }
        },
        { true, cudf::order::DESCENDING, cudf::null_order::AFTER }
    };
}

// single

template<typename T>
std::enable_if_t<std::is_floating_point<T>::value, test_case<T>>
single() {
    return test_case<T> {
        fixed_width_column_wrapper<T> ({ 7.309999942779541 }),
        {
            q_expect{ -1.0, 7.309999942779541, 7.309999942779541, 7.309999942779541, 7.309999942779541, 7.309999942779541, },
            q_expect{  0.0, 7.309999942779541, 7.309999942779541, 7.309999942779541, 7.309999942779541, 7.309999942779541, },
            q_expect{  1.0, 7.309999942779541, 7.309999942779541, 7.309999942779541, 7.309999942779541, 7.309999942779541, },
        }
    };
}

template<typename T>
std::enable_if_t<std::is_integral<T>::value and not cudf::is_boolean<T>(), test_case<T>>
single() {
    return test_case<T> {
        fixed_width_column_wrapper<T> ({ 1 }),
        {
            q_expect{ 0.7, 1, 1, 1, 1, 1 }
        }
    };
}

template<typename T>
std::enable_if_t<cudf::is_boolean<T>(), test_case<T>>
single() {
    return test_case<T> {
        fixed_width_column_wrapper<T> ({ 1 }),
        {
            q_expect{ 0.7, 1.0, 1.0, 1.0, 1.0, 1.0 }
        }
    };
}

// all_invalid

template<typename T>
std::enable_if_t<std::is_floating_point<T>::value, test_case<T>>
all_invalid() {
    return test_case<T> {
        fixed_width_column_wrapper<T> ({ 6.8, 0.15, 3.4, 4.17, 2.13, 1.11, -1.01, 0.8, 5.7 },
                                       { 0,      0,   0,    0,    0,    0,     0,   0,   0 }),
        {
            q_expect{ -1.0 },
            q_expect{  0.0 },
            q_expect{  0.5 },
            q_expect{  1.0 },
            q_expect{  2.0 }
        }
    };
}

template<typename T>
std::enable_if_t<std::is_integral<T>::value and not cudf::is_boolean<T>(), test_case<T>>
all_invalid() {
    return test_case<T> {
        fixed_width_column_wrapper<T> ({ 6, 0, 3, 4, 2, 1, -1, 1, 6 },
                                       { 0, 0, 0, 0, 0, 0,  0, 0, 0}),
        {
            q_expect{ 0.7 }
        }
    };
}

template<typename T>
std::enable_if_t<cudf::is_boolean<T>(), test_case<T>>
all_invalid() {
    return test_case<T> {
        fixed_width_column_wrapper<T> ({ 1, 0, 1, 1, 0, 1, 0, 1, 1 }, 
                                       { 0, 0, 0, 0, 0, 0, 0, 0, 0 }),
        {
            q_expect{ 0.7 }
        }
    };
}

// some invalid

template<typename T>
std::enable_if_t<std::is_same<T, double>::value, test_case<T>>
some_invalid() {
    T high = 0.16;
    T low = -1.024;
    T mid = -0.432;
    T lin = -0.432;
    return test_case<T> {
        fixed_width_column_wrapper<T> ({ 6.8, high, 3.4, 4.17, 2.13, 1.11, low, 0.8, 5.7 },
                                       { 0,      1,   0,    0,    0,    0,   1,   0,   0 }),
        {
            q_expect{ -1.0, low,  low,  low,  low,  low },
            q_expect{  0.0, low,  low,  low,  low,  low },
            q_expect{  0.5, high, low,  lin,  mid,  low },
            q_expect{  1.0, high, high, high, high, high },
            q_expect{  2.0, high, high, high, high, high }
        }
    };
}

template<typename T>
std::enable_if_t<std::is_same<T, float>::value, test_case<T>>
some_invalid() {
    T high = 0.16;
    T low = -1.024;
    double mid = -0.43200002610683441;
    double lin = -0.43200002610683441;
    return test_case<T> {
        fixed_width_column_wrapper<T> ({ 6.8, high, 3.4, 4.17, 2.13, 1.11, low, 0.8, 5.7 },
                                       { 0,      1,   0,    0,    0,    0,   1,   0,   0 }),
        {
            q_expect{ -1.0, low,  low,  low,  low,  low },
            q_expect{  0.0, low,  low,  low,  low,  low },
            q_expect{  0.5, high, low,  lin,  mid,  low },
            q_expect{  1.0, high, high, high, high, high },
            q_expect{  2.0, high, high, high, high, high }
        }
    };
}

template<typename T>
std::enable_if_t<std::is_integral<T>::value and not cudf::is_boolean<T>(), test_case<T>>
some_invalid() {
    return test_case<T> {
        fixed_width_column_wrapper<T> ({ 6, 0, 3, 4, 2, 1, -1, 1, 6 },
                                       { 0, 0, 1, 0, 0, 0,  0, 0, 1}),
        {
            q_expect{ 0.0, 3.0, 3.0, 3.0, 3.0, 3.0 },
            q_expect{ 0.5, 6.0, 3.0, 4.5, 4.5, 3.0 },
            q_expect{ 1.0, 6.0, 6.0, 6.0, 6.0, 6.0 }
        }
    };
}

template<typename T>
std::enable_if_t<cudf::is_boolean<T>(), test_case<T>>
some_invalid() {
    return test_case<T> {
        fixed_width_column_wrapper<T> ({ 1, 0, 1, 1, 0, 1, 0, 1, 1 },
                                       { 0, 0, 1, 0, 1, 0, 0, 0, 0}),
        {
            q_expect{ 0.0, 0.0, 0.0, 0.0, 0.0, 0.0 },
            q_expect{ 0.5, 1.0, 0.0, 0.5, 0.5, 0.0 },
            q_expect{ 1.0, 1.0, 1.0, 1.0, 1.0, 1.0 }
        }
    };
}

// unsorted

template<typename T>
std::enable_if_t<std::is_floating_point<T>::value, test_case<T>>
unsorted() {
    return test_case<T> {
        fixed_width_column_wrapper<T> ({ 6.8, 0.15, 3.4, 4.17, 2.13, 1.11, -1.00, 0.8, 5.7 }),
        {
            q_expect{ 0.0, -1.00, -1.00, -1.00, -1.00, -1.00 },
        }
    };
}

template<typename T>
std::enable_if_t<std::is_integral<T>::value and not cudf::is_boolean<T>(), test_case<T>>
unsorted() {
    return test_case<T> {
        fixed_width_column_wrapper<T> ({ 6, 0, 3, 4, 2, 1, -1, 1, 6 }),
        {
            q_expect{ 0.0, -1, -1, -1, -1, -1 }
        }
    };
}

template<typename T>
std::enable_if_t<cudf::is_boolean<T>(), test_case<T>>
unsorted() {
    return test_case<T> {
        fixed_width_column_wrapper<T> ({ 0, 0, 1, 1, 0, 1, 1, 0, 1 }),
        {
            q_expect{ 0.0, 0.0, 0.0, 0.0, 0.0, 0.0,}
        }
    };
}

} // namespace testdata

// =============================================================================
// ----- helper functions ------------------------------------------------------

template<typename T>
void test(testdata::test_case<T> test_case) {
    using namespace cudf::experimental;

    cudf::table_view in_table { { test_case.column } };

    for (auto & expected : test_case.expectations) {

        auto actual_higher = quantiles(in_table, expected.quantile, interpolation::HIGHER, { test_case.column_order });
        expect_scalars_equal(expected.higher, *actual_higher[0]);

        auto actual_lower = quantiles(in_table, expected.quantile, interpolation::LOWER, { test_case.column_order });
        expect_scalars_equal(expected.lower, *actual_lower[0]);

        auto actual_linear = quantiles(in_table, expected.quantile, interpolation::LINEAR, { test_case.column_order });
        expect_scalars_equal(expected.linear, *actual_linear[0]);

        auto actual_midpoint = quantiles(in_table, expected.quantile, interpolation::MIDPOINT, { test_case.column_order });
        expect_scalars_equal(expected.midpoint, *actual_midpoint[0]);

        auto actual_nearest = quantiles(in_table, expected.quantile, interpolation::NEAREST, { test_case.column_order });
        expect_scalars_equal(expected.nearest, *actual_nearest[0]);
    }
}

// =============================================================================
// ----- tests -----------------------------------------------------------------

template <typename T>
struct QuantilesTest : public BaseFixture {
};

using TestTypes = NumericTypes;
TYPED_TEST_CASE(QuantilesTest, TestTypes);

TYPED_TEST(QuantilesTest, TestEmpty)
{
    test(testdata::empty<TypeParam>());
}

TYPED_TEST(QuantilesTest, TestSingle)
{
    test(testdata::single<TypeParam>());
}

TYPED_TEST(QuantilesTest, TestSomeElementsInvalid)
{
    test(testdata::some_invalid<TypeParam>());
}

TYPED_TEST(QuantilesTest, TestAllElementsInvalid)
{
    test(testdata::all_invalid<TypeParam>());
}

TYPED_TEST(QuantilesTest, TestUnsorted)
{
    test(testdata::unsorted<TypeParam>());
}

TYPED_TEST(QuantilesTest, TestInterpolateCenter)
{
    test(testdata::interpolate_center<TypeParam>());
}

TYPED_TEST(QuantilesTest, TestInterpolateExtremaHigh)
{
    test(testdata::interpolate_extrema_high<TypeParam>());
}

TYPED_TEST(QuantilesTest, TestInterpolateExtremaLow)
{
    test(testdata::interpolate_extrema_low<TypeParam>());
}

TYPED_TEST(QuantilesTest, TestSortedAscendingNullBefore)
{
    test(testdata::sorted_ascending_null_before<TypeParam>());
}

TYPED_TEST(QuantilesTest, TestSortedDescendingNullAfter)
{
    test(testdata::sorted_descending_null_after<TypeParam>());
}

TYPED_TEST(QuantilesTest, TestMismatchedSortOrderCount)
{
    fixed_width_column_wrapper<TypeParam> a ({});
    fixed_width_column_wrapper<TypeParam> b ({});
    cudf::table_view input{{ a, b }};

    EXPECT_THROW(cudf::experimental::quantiles(input, 0, cudf::experimental::interpolation::LINEAR, { { false } }),
                 cudf::logic_error);
}

TYPED_TEST(QuantilesTest, TestEmptyTable)
{
    std::vector<cudf::column_view> input_columns = {};
    cudf::table_view input{ input_columns };
    auto q_values = cudf::experimental::quantiles(input, 0);
    EXPECT_EQ(0u, q_values.size());
}

TYPED_TEST(QuantilesTest, TestImplicitlyUnsortedInputs)
{
    auto a_val = std::numeric_limits<TypeParam>::lowest();
    auto b_val = std::numeric_limits<TypeParam>::max();

    fixed_width_column_wrapper<TypeParam> a ({ b_val, a_val });

    cudf::table_view input{{ a }};
    std::vector<std::unique_ptr<cudf::scalar>> q_values{};
    EXPECT_NO_THROW(q_values = cudf::experimental::quantiles(input, 0));
    auto q_expected = q_res(a_val);
    cudf::scalar * q_actual = q_values.at(0).get();
    expect_scalars_equal(q_expected, *q_actual);
}

template <typename T>
struct QuantilesUnsupportedTypesTest : public BaseFixture {
};

using UnsupportedTestTypes = RemoveIf<ContainedIn<TestTypes>, AllTypes>;
TYPED_TEST_CASE(QuantilesUnsupportedTypesTest, UnsupportedTestTypes);

TYPED_TEST(QuantilesUnsupportedTypesTest, TestZeroElements)
{
    fixed_width_column_wrapper<TypeParam> a ({ });
    cudf::table_view input{{ a }};

    EXPECT_THROW(cudf::experimental::quantiles(input, 0),
                 cudf::logic_error);
}

TYPED_TEST(QuantilesUnsupportedTypesTest, TestOneElements)
{
    fixed_width_column_wrapper<TypeParam> a ({ 0 });
    cudf::table_view input{{ a }};

    EXPECT_THROW(cudf::experimental::quantiles(input, 0),
                 cudf::logic_error);
}

TYPED_TEST(QuantilesUnsupportedTypesTest, TestMultipleElements)
{
    fixed_width_column_wrapper<TypeParam> a ({ 0, 1, 2 });
    cudf::table_view input{{ a }};

    EXPECT_THROW(cudf::experimental::quantiles(input, 0),
                 cudf::logic_error);
}

} // anonymous namespace
