#include <tests/strings/utilities.h>
#include <cudf/column/column_view.hpp>
#include <cudf/copying.hpp>
#include <cudf/detail/gather.cuh>
#include <cudf/detail/gather.hpp>
#include <cudf/table/table.hpp>
#include <cudf/table/table_view.hpp>
#include <tests/utilities/base_fixture.hpp>
#include <tests/utilities/column_utilities.hpp>
#include <tests/utilities/column_wrapper.hpp>
#include <tests/utilities/cudf_gtest.hpp>
#include <tests/utilities/table_utilities.hpp>
#include <tests/utilities/type_lists.hpp>

template <typename T>
class GatherTest : public cudf::test::BaseFixture {
};

TYPED_TEST_CASE(GatherTest, cudf::test::NumericTypes);

// This test exercises using different iterator types as gather map inputs
// to cudf::detail::gather -- device_vector and raw pointers.
TYPED_TEST(GatherTest, GatherDetailDeviceVectorTest)
{
  constexpr cudf::size_type source_size{1000};
  rmm::device_vector<cudf::size_type> gather_map(source_size);
  thrust::sequence(thrust::device, gather_map.begin(), gather_map.end());

  auto data = cudf::test::make_counting_transform_iterator(0, [](auto i) { return i; });
  cudf::test::fixed_width_column_wrapper<TypeParam> source_column(data, data + source_size);

  cudf::table_view source_table({source_column});

  // test with device vector iterators
  {
    std::unique_ptr<cudf::experimental::table> result =
      cudf::experimental::detail::gather(source_table, gather_map.begin(), gather_map.end());

    for (auto i = 0; i < source_table.num_columns(); ++i) {
      cudf::test::expect_columns_equal(source_table.column(i), result->view().column(i));
    }

    cudf::test::expect_tables_equal(source_table, result->view());
  }

  // test with raw pointers
  {
    std::unique_ptr<cudf::experimental::table> result = cudf::experimental::detail::gather(
      source_table, gather_map.data().get(), gather_map.data().get() + gather_map.size());

    for (auto i = 0; i < source_table.num_columns(); ++i) {
      cudf::test::expect_columns_equal(source_table.column(i), result->view().column(i));
    }

    cudf::test::expect_tables_equal(source_table, result->view());
  }
}

TYPED_TEST(GatherTest, GatherDetailInvalidIndexTest)
{
  constexpr cudf::size_type source_size{1000};

  auto data = cudf::test::make_counting_transform_iterator(0, [](auto i) { return i; });
  cudf::test::fixed_width_column_wrapper<TypeParam> source_column(data, data + source_size);
  auto gather_map_data =
    cudf::test::make_counting_transform_iterator(0, [](auto i) { return (i % 2) ? -1 : i; });
  cudf::test::fixed_width_column_wrapper<int32_t> gather_map(gather_map_data,
                                                             gather_map_data + (source_size * 2));

  cudf::table_view source_table({source_column});
  std::unique_ptr<cudf::experimental::table> result =
    cudf::experimental::detail::gather(source_table, gather_map, false, true);

  auto expect_data =
    cudf::test::make_counting_transform_iterator(0, [](auto i) { return (i % 2) ? 0 : i; });
  auto expect_valid = cudf::test::make_counting_transform_iterator(
    0, [](auto i) { return (i % 2) || (i >= source_size) ? 0 : 1; });
  cudf::test::fixed_width_column_wrapper<TypeParam> expect_column(
    expect_data, expect_data + (source_size * 2), expect_valid);

  for (auto i = 0; i < source_table.num_columns(); ++i) {
    cudf::test::expect_columns_equal(expect_column, result->view().column(i));
  }
}

TYPED_TEST(GatherTest, IdentityTest)
{
  constexpr cudf::size_type source_size{1000};

  auto data = cudf::test::make_counting_transform_iterator(0, [](auto i) { return i; });
  cudf::test::fixed_width_column_wrapper<TypeParam> source_column(data, data + source_size);
  cudf::test::fixed_width_column_wrapper<int32_t> gather_map(data, data + source_size);

  cudf::table_view source_table({source_column});

  std::unique_ptr<cudf::experimental::table> result =
    std::move(cudf::experimental::gather(source_table, gather_map));

  for (auto i = 0; i < source_table.num_columns(); ++i) {
    cudf::test::expect_columns_equal(source_table.column(i), result->view().column(i));
  }

  cudf::test::expect_tables_equal(source_table, result->view());
}

TYPED_TEST(GatherTest, ReverseIdentityTest)
{
  constexpr cudf::size_type source_size{1000};

  auto data = cudf::test::make_counting_transform_iterator(0, [](auto i) { return i; });
  auto reversed_data =
    cudf::test::make_counting_transform_iterator(0, [](auto i) { return source_size - 1 - i; });

  cudf::test::fixed_width_column_wrapper<TypeParam> source_column(data, data + source_size);
  cudf::test::fixed_width_column_wrapper<int32_t> gather_map(reversed_data,
                                                             reversed_data + source_size);

  cudf::table_view source_table({source_column});

  std::unique_ptr<cudf::experimental::table> result =
    std::move(cudf::experimental::gather(source_table, gather_map));
  cudf::test::fixed_width_column_wrapper<TypeParam> expect_column(reversed_data,
                                                                  reversed_data + source_size);

  for (auto i = 0; i < source_table.num_columns(); ++i) {
    cudf::test::expect_columns_equal(expect_column, result->view().column(i));
  }
}

TYPED_TEST(GatherTest, EveryOtherNullOdds)
{
  constexpr cudf::size_type source_size{1000};

  // Every other element is valid
  auto data     = cudf::test::make_counting_transform_iterator(0, [](auto i) { return i; });
  auto validity = cudf::test::make_counting_transform_iterator(0, [](auto i) { return i % 2; });

  cudf::test::fixed_width_column_wrapper<TypeParam> source_column(
    data, data + source_size, validity);

  // Gather odd-valued indices
  auto map_data = cudf::test::make_counting_transform_iterator(0, [](auto i) { return i * 2; });

  cudf::test::fixed_width_column_wrapper<int32_t> gather_map(map_data,
                                                             map_data + (source_size / 2));

  cudf::table_view source_table({source_column});

  std::unique_ptr<cudf::experimental::table> result =
    std::move(cudf::experimental::gather(source_table, gather_map));

  auto expect_data  = cudf::test::make_counting_transform_iterator(0, [](auto i) { return 0; });
  auto expect_valid = cudf::test::make_counting_transform_iterator(0, [](auto i) { return 0; });
  cudf::test::fixed_width_column_wrapper<TypeParam> expect_column(
    expect_data, expect_data + source_size / 2, expect_valid);

  for (auto i = 0; i < source_table.num_columns(); ++i) {
    cudf::test::expect_columns_equal(expect_column, result->view().column(i));
  }
}

TYPED_TEST(GatherTest, EveryOtherNullEvens)
{
  constexpr cudf::size_type source_size{1000};

  // Every other element is valid
  auto data     = cudf::test::make_counting_transform_iterator(0, [](auto i) { return i; });
  auto validity = cudf::test::make_counting_transform_iterator(0, [](auto i) { return i % 2; });

  cudf::test::fixed_width_column_wrapper<TypeParam> source_column(
    data, data + source_size, validity);

  // Gather even-valued indices
  auto map_data = cudf::test::make_counting_transform_iterator(0, [](auto i) { return i * 2 + 1; });

  cudf::test::fixed_width_column_wrapper<int32_t> gather_map(map_data,
                                                             map_data + (source_size / 2));

  cudf::table_view source_table({source_column});

  std::unique_ptr<cudf::experimental::table> result =
    std::move(cudf::experimental::gather(source_table, gather_map));

  auto expect_data =
    cudf::test::make_counting_transform_iterator(0, [](auto i) { return i * 2 + 1; });
  auto expect_valid = cudf::test::make_counting_transform_iterator(0, [](auto i) { return 1; });
  cudf::test::fixed_width_column_wrapper<TypeParam> expect_column(
    expect_data, expect_data + source_size / 2, expect_valid);

  for (auto i = 0; i < source_table.num_columns(); ++i) {
    cudf::test::expect_columns_equal(expect_column, result->view().column(i));
  }
}

TYPED_TEST(GatherTest, AllNull)
{
  constexpr cudf::size_type source_size{1000};

  // Every element is invalid
  auto data     = cudf::test::make_counting_transform_iterator(0, [](auto i) { return i; });
  auto validity = cudf::test::make_counting_transform_iterator(0, [](auto i) { return 0; });

  // Create a gather map that gathers to random locations
  std::vector<cudf::size_type> host_map_data(source_size);
  std::iota(host_map_data.begin(), host_map_data.end(), 0);
  std::mt19937 g(0);
  std::shuffle(host_map_data.begin(), host_map_data.end(), g);
  thrust::device_vector<cudf::size_type> map_data(host_map_data);

  cudf::test::fixed_width_column_wrapper<TypeParam> source_column{
    data, data + source_size, validity};
  cudf::test::fixed_width_column_wrapper<int32_t> gather_map(map_data.begin(), map_data.end());

  cudf::table_view source_table({source_column});

  std::unique_ptr<cudf::experimental::table> result =
    std::move(cudf::experimental::gather(source_table, gather_map));

  // Check that the result is also all invalid
  cudf::test::expect_tables_equal(source_table, result->view());
}

TYPED_TEST(GatherTest, MultiColReverseIdentityTest)
{
  constexpr cudf::size_type source_size{1000};

  constexpr cudf::size_type n_cols = 3;

  auto data = cudf::test::make_counting_transform_iterator(0, [](auto i) { return i; });
  auto reversed_data =
    cudf::test::make_counting_transform_iterator(0, [](auto i) { return source_size - 1 - i; });

  std::vector<cudf::test::fixed_width_column_wrapper<TypeParam>> source_column_wrappers;
  std::vector<cudf::column_view> source_columns;

  for (int i = 0; i < n_cols; ++i) {
    source_column_wrappers.push_back(
      cudf::test::fixed_width_column_wrapper<TypeParam>(data, data + source_size));
    source_columns.push_back(source_column_wrappers[i]);
  }

  cudf::test::fixed_width_column_wrapper<int32_t> gather_map(reversed_data,
                                                             reversed_data + source_size);

  cudf::table_view source_table{source_columns};

  std::unique_ptr<cudf::experimental::table> result =
    std::move(cudf::experimental::gather(source_table, gather_map));

  cudf::test::fixed_width_column_wrapper<TypeParam> expect_column(reversed_data,
                                                                  reversed_data + source_size);

  for (auto i = 0; i < source_table.num_columns(); ++i) {
    cudf::test::expect_columns_equal(expect_column, result->view().column(i));
  }
}

TYPED_TEST(GatherTest, MultiColNulls)
{
  constexpr cudf::size_type source_size{1000};

  static_assert(0 == source_size % 2, "Size of source data must be a multiple of 2.");

  constexpr cudf::size_type n_cols = 3;

  auto data     = cudf::test::make_counting_transform_iterator(0, [](auto i) { return i; });
  auto validity = cudf::test::make_counting_transform_iterator(0, [](auto i) { return i % 2; });

  std::vector<cudf::test::fixed_width_column_wrapper<TypeParam>> source_column_wrappers;
  std::vector<cudf::column_view> source_columns;

  for (int i = 0; i < n_cols; ++i) {
    source_column_wrappers.push_back(
      cudf::test::fixed_width_column_wrapper<TypeParam>(data, data + source_size, validity));
    source_columns.push_back(source_column_wrappers[i]);
  }

  auto reversed_data =
    cudf::test::make_counting_transform_iterator(0, [](auto i) { return source_size - 1 - i; });

  cudf::test::fixed_width_column_wrapper<int32_t> gather_map(reversed_data,
                                                             reversed_data + source_size);

  cudf::table_view source_table{source_columns};

  std::unique_ptr<cudf::experimental::table> result =
    std::move(cudf::experimental::gather(source_table, gather_map));

  // Expected data
  auto expect_data =
    cudf::test::make_counting_transform_iterator(0, [](auto i) { return source_size - i - 1; });
  auto expect_valid =
    cudf::test::make_counting_transform_iterator(0, [](auto i) { return (i + 1) % 2; });

  cudf::test::fixed_width_column_wrapper<TypeParam> expect_column(
    expect_data, expect_data + source_size, expect_valid);

  for (auto i = 0; i < source_table.num_columns(); ++i) {
    cudf::test::expect_columns_equal(expect_column, result->view().column(i));
  }
}

class GatherTestStr : public cudf::test::BaseFixture {
};

TEST_F(GatherTestStr, StringColumn)
{
  cudf::test::fixed_width_column_wrapper<int16_t> col1{{1, 2, 3, 4, 5, 6}, {1, 1, 0, 1, 0, 1}};
  cudf::test::strings_column_wrapper col2{{"This", "is", "not", "a", "string", "type"},
                                          {1, 1, 1, 1, 1, 0}};
  cudf::table_view source_table{{col1, col2}};

  cudf::test::fixed_width_column_wrapper<int16_t> gather_map{{0, 1, 3, 4}};

  cudf::test::fixed_width_column_wrapper<int16_t> exp_col1{{1, 2, 4, 5}, {1, 1, 1, 0}};
  cudf::test::strings_column_wrapper exp_col2{{"This", "is", "a", "string"}, {1, 1, 1, 1}};
  cudf::table_view expected{{exp_col1, exp_col2}};

  auto got = cudf::experimental::gather(source_table, gather_map);

  cudf::test::expect_tables_equal(expected, got->view());
}

TEST_F(GatherTestStr, Gather)
{
  std::vector<const char*> h_strings{"eee", "bb", "", "aa", "bbb", "ééé"};
  cudf::test::strings_column_wrapper strings(h_strings.begin(), h_strings.end());
  cudf::table_view source_table({strings});

  std::vector<int32_t> h_map{4, 1, 5, 2, 7};
  cudf::test::fixed_width_column_wrapper<int32_t> gather_map(h_map.begin(), h_map.end());
  auto results = cudf::experimental::detail::gather(source_table, gather_map, false, true);

  std::vector<const char*> h_expected;
  std::vector<int32_t> expected_validity;
  for (auto itr = h_map.begin(); itr != h_map.end(); ++itr) {
    auto index = *itr;
    if ((0 <= index) && (index < static_cast<decltype(index)>(h_strings.size()))) {
      h_expected.push_back(h_strings[index]);
      expected_validity.push_back(1);
    } else {
      h_expected.push_back("");
      expected_validity.push_back(0);
    }
  }
  cudf::test::strings_column_wrapper expected(
    h_expected.begin(), h_expected.end(), expected_validity.begin());
  cudf::test::expect_columns_equal(results->view().column(0), expected);
}

TEST_F(GatherTestStr, GatherIgnoreOutOfBounds)
{
  std::vector<const char*> h_strings{"eee", "bb", "", "aa", "bbb", "ééé"};
  cudf::test::strings_column_wrapper strings(h_strings.begin(), h_strings.end());
  cudf::table_view source_table({strings});

  std::vector<int32_t> h_map{3, 4, 0, 0};
  cudf::test::fixed_width_column_wrapper<int32_t> gather_map(h_map.begin(), h_map.end());
  auto results = cudf::experimental::detail::gather(source_table, gather_map, false, true);

  std::vector<const char*> h_expected;
  std::vector<int32_t> expected_validity;
  for (auto itr = h_map.begin(); itr != h_map.end(); ++itr) {
    h_expected.push_back(h_strings[*itr]);
    expected_validity.push_back(1);
  }
  cudf::test::strings_column_wrapper expected(
    h_expected.begin(), h_expected.end(), expected_validity.begin());
  cudf::test::expect_columns_equal(results->view().column(0), expected);
}

TEST_F(GatherTestStr, GatherZeroSizeStringsColumn)
{
  cudf::column_view zero_size_strings_column(cudf::data_type{cudf::STRING}, 0, nullptr, nullptr, 0);
  rmm::device_vector<cudf::size_type> gather_map{};
  auto results = cudf::experimental::detail::gather(
    cudf::table_view({zero_size_strings_column}), gather_map.begin(), gather_map.end(), true);
  cudf::test::expect_strings_empty(results->get_column(0).view());
}
