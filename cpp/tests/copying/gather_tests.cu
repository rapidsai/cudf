/*
 * Copyright (c) 2019, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <cudf/copying.hpp>
#include <cudf/legacy/table.hpp>

#include <gmock/gmock.h>
#include <gtest/gtest.h>
#include <tests/utilities/column_wrapper.cuh>
#include "tests/utilities/compare_column_wrappers.cuh"
#include <tests/utilities/cudf_test_fixtures.h>
#include <tests/utilities/cudf_test_utils.cuh>
#include <tests/utilities/compare_column_wrappers.cuh>
#include <cudf/types.hpp>
#include <utilities/wrapper_types.hpp>

#include <random>

template <typename T>
struct GatherTest : GdfTest {};

using test_types =
    ::testing::Types<int8_t, int16_t, int32_t, int64_t, float, double, cudf::bool8>;
TYPED_TEST_CASE(GatherTest, test_types);

TYPED_TEST(GatherTest, DtypeMistach){
  constexpr gdf_size_type source_size{1000};
  constexpr gdf_size_type destination_size{1000};

  cudf::test::column_wrapper<int32_t> source{source_size};
  cudf::test::column_wrapper<float> destination{destination_size};

  gdf_column * raw_source = source.get();
  gdf_column * raw_destination = destination.get();

  cudf::table source_table{&raw_source, 1};
  cudf::table destination_table{&raw_destination, 1};

  rmm::device_vector<gdf_index_type> gather_map(source_size);

  EXPECT_THROW(cudf::gather(&source_table, gather_map.data().get(),
                             &destination_table), cudf::logic_error);
}

TYPED_TEST(GatherTest, DestMissingValid){
  constexpr gdf_size_type source_size{1000};
  constexpr gdf_size_type destination_size{1000};

  cudf::test::column_wrapper<TypeParam> source(source_size, true);
  cudf::test::column_wrapper<TypeParam> destination(destination_size, false);

  gdf_column * raw_source = source.get();
  gdf_column * raw_destination = destination.get();

  cudf::table source_table{&raw_source, 1};
  cudf::table destination_table{&raw_destination, 1};

  rmm::device_vector<gdf_index_type> gather_map(source_size);

  EXPECT_THROW(cudf::gather(&source_table, gather_map.data().get(),
                             &destination_table), cudf::logic_error);
}

TYPED_TEST(GatherTest, NumColumnsMismatch){
  constexpr gdf_size_type source_size{1000};
  constexpr gdf_size_type destination_size{1000};

  cudf::test::column_wrapper<TypeParam> source0(source_size, true);
  cudf::test::column_wrapper<TypeParam> source1(source_size, true);
  cudf::test::column_wrapper<TypeParam> destination(destination_size, false);

  std::vector<gdf_column*> source_cols{source0.get(), source1.get()};

  gdf_column * raw_destination = destination.get();

  cudf::table source_table{source_cols.data(), 2};
  cudf::table destination_table{&raw_destination, 1};

  rmm::device_vector<gdf_index_type> gather_map(source_size);

  EXPECT_THROW(cudf::gather(&source_table, gather_map.data().get(),
                             &destination_table), cudf::logic_error);
}

TYPED_TEST(GatherTest, IdentityTest) {
  constexpr gdf_size_type source_size{1000};
  constexpr gdf_size_type destination_size{1000};

  cudf::test::column_wrapper<TypeParam> source_column{
      source_size, [](gdf_index_type row) { return static_cast<TypeParam>(row); },
      [](gdf_index_type row) { return true; }};

  thrust::device_vector<gdf_index_type> gather_map(destination_size);
  thrust::sequence(gather_map.begin(), gather_map.end());

  cudf::test::column_wrapper<TypeParam> destination_column(destination_size,
                                                           true);

  gdf_column* raw_source = source_column.get();
  gdf_column* raw_destination = destination_column.get();

  cudf::table source_table{&raw_source, 1};
  cudf::table destination_table{&raw_destination, 1};

  EXPECT_NO_THROW(
      cudf::gather(&source_table, gather_map.data().get(), &destination_table));

  expect_columns_are_equal<TypeParam>(source_column, destination_column);
}

TYPED_TEST(GatherTest, ReverseIdentityTest) {
  constexpr gdf_size_type source_size{1000};
  constexpr gdf_size_type destination_size{1000};

  static_assert(source_size == destination_size,
                "Source and destination columns must be the same size.");

  cudf::test::column_wrapper<TypeParam> source_column{
      source_size, [](gdf_index_type row) { return static_cast<TypeParam>(row); },
      [](gdf_index_type row) { return true; }};

  // Create gather_map that reverses order of source_column
  std::vector<gdf_index_type> host_gather_map(source_size);
  std::iota(host_gather_map.begin(), host_gather_map.end(), 0);
  std::reverse(host_gather_map.begin(), host_gather_map.end());
  thrust::device_vector<gdf_index_type> gather_map(host_gather_map);

  cudf::test::column_wrapper<TypeParam> destination_column(destination_size,
                                                           true);

  gdf_column* raw_source = source_column.get();
  gdf_column* raw_destination = destination_column.get();

  cudf::table source_table{&raw_source, 1};
  cudf::table destination_table{&raw_destination, 1};

  EXPECT_NO_THROW(
      cudf::gather(&source_table, gather_map.data().get(), &destination_table));

  // Expected result is the reversal of the source column
  std::vector<TypeParam> expected_data;
  std::vector<gdf_valid_type> expected_bitmask;
  std::tie(expected_data, expected_bitmask) = source_column.to_host();
  std::reverse(expected_data.begin(), expected_data.end());

  // Copy result of destination column to host
  std::vector<TypeParam> result_data;
  std::vector<gdf_valid_type> result_bitmask;
  std::tie(result_data, result_bitmask) = destination_column.to_host();

  auto print_all_unequal_pairs { true };
  expect_column_values_are_equal<TypeParam>(
      destination_size, expected_data.data(), nullptr, "Expected",
      result_data.data(), nullptr, "Actual",
      print_all_unequal_pairs);

  for (gdf_index_type i = 0; i < destination_size; i++) {
    EXPECT_TRUE(gdf_is_valid(result_bitmask.data(), i))
        << "Value at index " << i << " should be non-null!\n";
  }
}

TYPED_TEST(GatherTest, AllNull) {
  constexpr gdf_size_type source_size{1000};
  constexpr gdf_size_type destination_size{1000};

  // source column has all null values
  cudf::test::column_wrapper<TypeParam> source_column{
      source_size, [](gdf_index_type row) { return static_cast<TypeParam>(row); },
      [](gdf_index_type row) { return false; }};

  // Create gather_map that gathers to random locations
  std::vector<gdf_index_type> host_gather_map(source_size);
  std::iota(host_gather_map.begin(), host_gather_map.end(), 0);
  std::mt19937 g(0);
  std::shuffle(host_gather_map.begin(), host_gather_map.end(), g);
  thrust::device_vector<gdf_index_type> gather_map(host_gather_map);

  cudf::test::column_wrapper<TypeParam> destination_column(destination_size,
                                                           true);

  gdf_column* raw_source = source_column.get();
  gdf_column* raw_destination = destination_column.get();

  cudf::table source_table{&raw_source, 1};
  cudf::table destination_table{&raw_destination, 1};

  EXPECT_NO_THROW(
      cudf::gather(&source_table, gather_map.data().get(), &destination_table));

  // Copy result of destination column to host
  std::vector<TypeParam> result_data;
  std::vector<gdf_valid_type> result_bitmask;
  std::tie(result_data, result_bitmask) = destination_column.to_host();

  // All values of result should be null
  for (gdf_index_type i = 0; i < destination_size; i++) {
    EXPECT_FALSE(gdf_is_valid(result_bitmask.data(), i))
        << "Value at index " << i << " should be null!\n";
  }
}

TYPED_TEST(GatherTest, EveryOtherNull) {
  constexpr gdf_size_type source_size{1234};
  constexpr gdf_size_type destination_size{source_size};

  static_assert(0 == source_size % 2,
                "Size of source data must be a multiple of 2.");
  static_assert(source_size == destination_size,
                "Source and destination columns must be equal size.");

  // elements with even indices are null
  cudf::test::column_wrapper<TypeParam> source_column{
      source_size, [](gdf_index_type row) { return static_cast<TypeParam>(row); },
      [](gdf_index_type row) { return row % 2; }};

  // Gather null values to the last half of the destination column
  std::vector<gdf_index_type> host_gather_map(source_size);
  for (gdf_size_type i = 0; i < destination_size / 2; ++i) {
    host_gather_map[i] = i * 2 + 1;
    host_gather_map[destination_size / 2 + i] = i * 2;
  }

  thrust::device_vector<gdf_index_type> gather_map(host_gather_map);

  cudf::test::column_wrapper<TypeParam> destination_column(destination_size,
                                                           true);

  gdf_column* raw_source = source_column.get();
  gdf_column* raw_destination = destination_column.get();

  cudf::table source_table{&raw_source, 1};
  cudf::table destination_table{&raw_destination, 1};

  EXPECT_NO_THROW(
      cudf::gather(&source_table, gather_map.data().get(), &destination_table));

  // Copy result of destination column to host
  std::vector<TypeParam> result_data;
  std::vector<gdf_valid_type> result_bitmask;
  std::tie(result_data, result_bitmask) = destination_column.to_host();

  for (gdf_index_type i = 0; i < destination_size; i++) {
    // The first half of the destination column should be all valid
    // and values should be 1, 3, 5, 7, etc.
    if (i < destination_size / 2) {
      EXPECT_TRUE(gdf_is_valid(result_bitmask.data(), i))
          << "Value at index " << i << " should be non-null!\n";
      EXPECT_EQ(static_cast<TypeParam>(i * 2 + 1), result_data[i]);
    } else {
      // The last half of the destination column should be all null
      EXPECT_FALSE(gdf_is_valid(result_bitmask.data(), i))
          << "Value at index " << i << " should be null!\n";
    }
  }
}
