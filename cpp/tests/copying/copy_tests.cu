/*
 * Copyright (c) 2019, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <tests/utilities/cudf_gtest.hpp>
#include <cudf/copying.hpp>
#include <cudf/detail/copy_if_else.cuh>
#include <tests/utilities/base_fixture.hpp>
#include <tests/utilities/type_lists.hpp>
#include <tests/utilities/legacy/cudf_test_utils.cuh>

#include <tests/utilities/column_wrapper.hpp>
#include <tests/utilities/column_utilities.hpp>

#include <cudf/column/column.hpp>
#include <cudf/scalar/scalar.hpp>
#include <cudf/column/column_device_view.cuh>

#include <iterator/legacy/iterator.cuh>

template <typename T>
struct CopyTest : public cudf::test::BaseFixture {};

TYPED_TEST_CASE(CopyTest, cudf::test::FixedWidthTypes);

// to keep names shorter
#define wrapper cudf::test::fixed_width_column_wrapper
using bool_wrapper = wrapper<cudf::experimental::bool8>;

TYPED_TEST(CopyTest, CopyIfElseTestShort) 
{ 
   using T = TypeParam;

   // short one. < 1 warp/bitmask length
   int num_els = 4;

   bool mask[]    = { 1, 0, 0, 0 };
   bool_wrapper mask_w(mask, mask + num_els);

   T lhs[]        = { 5, 5, 5, 5 }; 
   bool lhs_v[]   = { 1, 1, 1, 1 };
   wrapper<T> lhs_w(lhs, lhs + num_els, lhs_v);

   T rhs[]        = { 6, 6, 6, 6 };
   bool rhs_v[]   = { 1, 1, 1, 1 };
   wrapper<T> rhs_w(rhs, rhs + num_els, rhs_v);
   
   T expected[]   = { 5, 6, 6, 6 };
   // bool exp_v[]   = { 1, 1, 1, 1 };
   wrapper<T> expected_w(expected, expected + num_els);

   auto out = cudf::experimental::copy_if_else(lhs_w, rhs_w, mask_w);
   cudf::test::expect_columns_equal(out->view(), expected_w);
}

TYPED_TEST(CopyTest, CopyIfElseTestManyNulls) 
{ 
   using T = TypeParam;

   // bunch of nulls in output, non-aligned # of elements
   int num_els = 7;

   bool mask[]    = { 1, 0, 0, 0, 0, 0, 1 };
   bool_wrapper mask_w(mask, mask + num_els);

   T lhs[]        = { 5, 5, 5, 5, 5, 5, 5 }; 
   bool lhs_v[]   = { 1, 1, 1, 1, 1, 1, 1 };
   wrapper<T> lhs_w(lhs, lhs + num_els, lhs_v);

   T rhs[]        = { 6, 6, 6, 6, 6, 6, 6 };
   bool rhs_v[]   = { 1, 0, 0, 0, 0, 0, 1 };
   wrapper<T> rhs_w(rhs, rhs + num_els, rhs_v);
   
   T expected[]   = { 5, 6, 6, 6, 6, 6, 5 };
   bool exp_v[]   = { 1, 0, 0, 0, 0, 0, 1 };
   wrapper<T> expected_w(expected, expected + num_els, exp_v);

   auto out = cudf::experimental::copy_if_else(lhs_w, rhs_w, mask_w);      
   cudf::test::expect_columns_equal(out->view(), expected_w);   
}

struct copy_if_else_tiny_grid_functor {
   template <typename T, typename Filter, std::enable_if_t<cudf::is_fixed_width<T>()>* = nullptr>
   std::unique_ptr<cudf::column> operator()(cudf::column_view const& lhs,
                                          cudf::column_view const& rhs,
                                          Filter filter,
                                          rmm::mr::device_memory_resource *mr,
                                          hipStream_t stream)
   {
      // output
      std::unique_ptr<cudf::column> out = cudf::experimental::allocate_like(lhs, lhs.size(), cudf::experimental::mask_allocation_policy::RETAIN, mr);

      // device views      
      auto lhs_iter = cudf::make_pair_iterator<false, T>((T*)lhs.begin<T>(), static_cast<cudf::bitmask_type*>(nullptr));
      auto rhs_iter = cudf::make_pair_iterator<false, T>((T*)rhs.begin<T>(), static_cast<cudf::bitmask_type*>(nullptr));
      auto out_dv = cudf::mutable_column_device_view::create(*out);
             
      // call the kernel with an artificially small grid
      cudf::experimental::detail::copy_if_else_kernel<32, T, decltype(lhs_iter), decltype(rhs_iter), Filter, false><<<1, 32, 0, stream>>>(
         lhs_iter, rhs_iter, filter, *out_dv, nullptr);

      return out;
   }

   template <typename T, typename Filter, std::enable_if_t<not cudf::is_fixed_width<T>()>* = nullptr>
   std::unique_ptr<cudf::column> operator()(cudf::column_view const& lhs,
                                          cudf::column_view const& rhs,
                                          Filter filter,
                                          rmm::mr::device_memory_resource *mr,
                                          hipStream_t stream)
   {
      CUDF_FAIL("Unexpected test execution");
   }
};

std::unique_ptr<cudf::column> tiny_grid_launch(cudf::column_view const& lhs, cudf::column_view const& rhs, cudf::column_view const& boolean_mask)
{
   auto bool_mask_device_p = cudf::column_device_view::create(boolean_mask);
   cudf::column_device_view bool_mask_device = *bool_mask_device_p;
   auto filter = [bool_mask_device] __device__ (cudf::size_type i) { return bool_mask_device.element<cudf::experimental::bool8>(i); };
   return cudf::experimental::type_dispatcher(lhs.type(),
                                             copy_if_else_tiny_grid_functor{},
                                             lhs,
                                             rhs,
                                             filter,
                                             rmm::mr::get_default_resource(),
                                             (hipStream_t)0);
}

TYPED_TEST(CopyTest, CopyIfElseTestTinyGrid) 
{  
   using T = TypeParam;

   // make sure we span at least 2 warps      
   int num_els = 64;

   bool mask[]    = { 1, 0, 1, 0, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 0, 0, 0, 0, 1, 1, 1, 1, 1, 1, 1, 1, 1, 
                     0, 0, 0, 0, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1 };   
   bool_wrapper mask_w(mask, mask + num_els);

   T lhs[]        = { 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 
                     5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5 };   
   wrapper<T> lhs_w(lhs, lhs + num_els);

   T rhs[]        = { 6, 6, 6, 6, 6, 6, 6, 6, 6, 6, 6, 6, 6, 6, 6, 6, 6, 6, 6, 6, 6, 6, 6, 6, 6, 6, 6, 6, 6, 6, 6, 6,
                     6, 6, 6, 6, 6, 6, 6, 6, 6, 6, 6, 6, 6, 6, 6, 6, 6, 6, 6, 6, 6, 6, 6, 6, 6, 6, 6, 6, 6, 6, 6, 6 };
   wrapper<T> rhs_w(rhs, rhs + num_els);
   
   T expected[]   = { 5, 6, 5, 6, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 6, 6, 6, 6, 5, 5, 5, 5, 5, 5, 5, 5, 5, 
                     6, 6, 6, 6, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5 };   
   wrapper<T> expected_w(expected, expected + num_els);

   auto out = tiny_grid_launch(lhs_w, rhs_w, mask_w);
     
   cudf::test::expect_columns_equal(out->view(), expected_w);   
}

TYPED_TEST(CopyTest, CopyIfElseTestLong) 
{  
   using T = TypeParam;

   // make sure we span at least 2 warps      
   int num_els = 64;

   bool mask[]    = { 1, 0, 1, 0, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 0, 0, 0, 0, 1, 1, 1, 1, 1, 1, 1, 1, 1, 
                     0, 0, 0, 0, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1 };   
   bool_wrapper mask_w(mask, mask + num_els);

   T lhs[]        = { 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 
                     5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5 };
   bool lhs_v[]   = { 1, 1, 1, 1, 0, 0, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 
                     1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1 };   
   wrapper<T> lhs_w(lhs, lhs + num_els, lhs_v);

   T rhs[]        = { 6, 6, 6, 6, 6, 6, 6, 6, 6, 6, 6, 6, 6, 6, 6, 6, 6, 6, 6, 6, 6, 6, 6, 6, 6, 6, 6, 6, 6, 6, 6, 6,
                     6, 6, 6, 6, 6, 6, 6, 6, 6, 6, 6, 6, 6, 6, 6, 6, 6, 6, 6, 6, 6, 6, 6, 6, 6, 6, 6, 6, 6, 6, 6, 6 };
   bool rhs_v[]   = { 1, 1, 1, 1, 1, 1, 0, 0, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 
                     1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1 };                      
   wrapper<T> rhs_w(rhs, rhs + num_els, rhs_v);
   
   T expected[]   = { 5, 6, 5, 6, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 6, 6, 6, 6, 5, 5, 5, 5, 5, 5, 5, 5, 5, 
                     6, 6, 6, 6, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5, 5 };
   bool exp_v[]   = { 1, 1, 1, 1, 0, 0, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 
                     1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1 };   
   wrapper<T> expected_w(expected, expected + num_els, exp_v);

   auto out = cudf::experimental::copy_if_else(lhs_w, rhs_w, mask_w);      
   cudf::test::expect_columns_equal(out->view(), expected_w);   
}

TYPED_TEST(CopyTest, CopyIfElseTestEmptyInputs) 
{ 
   using T = TypeParam;
         
   int num_els = 0;
   
   bool mask[]    = {};
   bool_wrapper mask_w(mask, mask + num_els);

   T lhs[]        = {};
   wrapper<T> lhs_w(lhs, lhs + num_els);

   T rhs[]        = {};
   wrapper<T> rhs_w(rhs, rhs + num_els);
   
   T expected[]   = {};
   wrapper<T> expected_w(expected, expected + num_els);

   auto out = cudf::experimental::copy_if_else(lhs_w, rhs_w, mask_w);      
   cudf::test::expect_columns_equal(out->view(), expected_w);
}

TYPED_TEST(CopyTest, CopyIfElseMixedInputValidity)
{ 
   using T = TypeParam;   
         
   int num_els = 4;

   bool mask[]    = { 1, 0, 1, 1 };
   bool_wrapper mask_w(mask, mask + num_els);

   T lhs[]        = { 5, 5, 5, 5 }; 
   bool lhs_m[]   = { 1, 1, 1, 0 };
   wrapper<T> lhs_w(lhs, lhs + num_els, lhs_m);

   T rhs[]        = { 6, 6, 6, 6 };
   bool rhs_m[]   = { 1, 0, 1, 1 };
   wrapper<T> rhs_w(rhs, rhs + num_els, rhs_m);

   T expected[]   = { 5, 6, 5, 5 };
   bool exp_m[]   = { 1, 0, 1, 0 };
   wrapper<T> expected_w(expected, expected + num_els, exp_m);

   auto out = cudf::experimental::copy_if_else(lhs_w, rhs_w, mask_w);
   cudf::test::expect_columns_equal(out->view(), expected_w);
}

TYPED_TEST(CopyTest, CopyIfElseMixedInputValidity2)
{ 
   using T = TypeParam;

   int num_els = 4;

   bool mask[]    = { 1, 0, 1, 1 };
   bool_wrapper mask_w(mask, mask + num_els);

   T lhs[]        = { 5, 5, 5, 5 }; 
   bool lhs_m[]   = { 1, 1, 1, 0 };
   wrapper<T> lhs_w(lhs, lhs + num_els, lhs_m);

   T rhs[]        = { 6, 6, 6, 6 };
   wrapper<T> rhs_w(rhs, rhs + num_els);

   T expected[]   = { 5, 6, 5, 5 };
   bool exp_m[]   = { 1, 1, 1, 0 };
   wrapper<T> expected_w(expected, expected + num_els, exp_m);

   auto out = cudf::experimental::copy_if_else(lhs_w, rhs_w, mask_w); 
   cudf::test::expect_columns_equal(out->view(), expected_w); 
}

TYPED_TEST(CopyTest, CopyIfElseMixedInputValidity3)
{ 
   using T = TypeParam;

   int num_els = 4;

   bool mask[]    = { 1, 0, 1, 1 };
   bool_wrapper mask_w(mask, mask + num_els);

   T lhs[]        = { 5, 5, 5, 5 };
   wrapper<T> lhs_w(lhs, lhs + num_els);

   T rhs[]        = { 6, 6, 6, 6 };
   bool rhs_m[]   = { 1, 0, 1, 1 };
   wrapper<T> rhs_w(rhs, rhs + num_els, rhs_m);

   T expected[]   = { 5, 6, 5, 5 };
   bool exp_m[]   = { 1, 0, 1, 1 };
   wrapper<T> expected_w(expected, expected + num_els, exp_m);

   auto out = cudf::experimental::copy_if_else(lhs_w, rhs_w, mask_w);
   cudf::test::expect_columns_equal(out->view(), expected_w);
}

TYPED_TEST(CopyTest, CopyIfElseMixedInputValidity4)
{ 
   using T = TypeParam;

   int num_els = 4;

   bool mask[]    = { 1, 0, 1, 1 };
   bool_wrapper mask_w(mask, mask + num_els);

   T lhs[]        = { 5, 5, 5, 5 };
   wrapper<T> lhs_w(lhs, lhs + num_els);

   T rhs[]        = { 6, 6, 6, 6 };
   wrapper<T> rhs_w(rhs, rhs + num_els);

   T expected[]   = { 5, 6, 5, 5 };
   wrapper<T> expected_w(expected, expected + num_els);

   auto out = cudf::experimental::copy_if_else(lhs_w, rhs_w, mask_w);
   cudf::test::expect_columns_equal(out->view(), expected_w);
}

TYPED_TEST(CopyTest, CopyIfElseBadInputLength)
{ 
   using T = TypeParam;   
         
   int num_els = 4;

   // mask length mismatch
   {
      bool mask[]    = { 1, 1, 1, 1 };
      bool_wrapper mask_w(mask, mask + 3);

      T lhs[]        = { 5, 5, 5, 5 };
      wrapper<T> lhs_w(lhs, lhs + num_els, mask);

      T rhs[]        = { 6, 6, 6, 6 };
      wrapper<T> rhs_w(rhs, rhs + num_els, mask);

      EXPECT_THROW(  cudf::experimental::copy_if_else(lhs_w, rhs_w, mask_w),
                     cudf::logic_error);
   }

   // column length mismatch
   {
      bool mask[]    = { 1, 1, 1, 1 };
      bool_wrapper mask_w(mask, mask + num_els);

      T lhs[]        = { 5, 5, 5 };
      wrapper<T> lhs_w(lhs, lhs + 3, mask);

      T rhs[]        = { 6, 6, 6, 6 };
      wrapper<T> rhs_w(rhs, rhs + num_els, mask);

      EXPECT_THROW(  cudf::experimental::copy_if_else(lhs_w, rhs_w, mask_w), 
                     cudf::logic_error);
   }
}

template <typename T>
struct CopyTestNumeric : public cudf::test::BaseFixture {};
TYPED_TEST_CASE(CopyTestNumeric, cudf::test::NumericTypes);

TYPED_TEST(CopyTestNumeric, CopyIfElseTestScalarColumn) 
{ 
   using T = TypeParam;
   
   int num_els = 4;

   bool mask[]    = { 1, 0, 0, 1 };
   bool_wrapper mask_w(mask, mask + num_els);
   
   cudf::numeric_scalar<T> lhs_w(5);

   T rhs[]        = { 6, 6, 6, 6 };
   bool rhs_v[]   = { 1, 0, 1, 1 };
   wrapper<T> rhs_w(rhs, rhs + num_els, rhs_v);
   
   T expected[]   = { 5, 6, 6, 5 };   
   wrapper<T> expected_w(expected, expected + num_els, rhs_v);

   auto out = cudf::experimental::copy_if_else(lhs_w, rhs_w, mask_w);
   cudf::test::expect_columns_equal(out->view(), expected_w);
}

TYPED_TEST(CopyTestNumeric, CopyIfElseTestColumnScalar) 
{ 
   using T = TypeParam;
   
   int num_els = 4;

   bool mask[]    = { 1, 0, 0, 1 };
   bool_wrapper mask_w(mask, mask + num_els);
   
   T lhs[]        = { 5, 5, 5, 5 };
   bool lhs_v[]   = { 0, 1, 1, 1 };
   wrapper<T> lhs_w(lhs, lhs + num_els, lhs_v);

   cudf::numeric_scalar<T> rhs_w(6);
   
   T expected[]   = { 5, 6, 6, 5 };   
   wrapper<T> expected_w(expected, expected + num_els, lhs_v);

   auto out = cudf::experimental::copy_if_else(lhs_w, rhs_w, mask_w);
   cudf::test::expect_columns_equal(out->view(), expected_w);
}

TYPED_TEST(CopyTestNumeric, CopyIfElseTestScalarScalar) 
{ 
   using T = TypeParam;
   
   int num_els = 4;   

   bool mask[]    = { 1, 0, 0, 1 };
   bool_wrapper mask_w(mask, mask + num_els);

   cudf::numeric_scalar<T> lhs_w(5);

   cudf::numeric_scalar<T> rhs_w(6, false);
   
   T expected[]   = { 5, 6, 6, 5 };   
   wrapper<T> expected_w(expected, expected + num_els, mask);

   auto out = cudf::experimental::copy_if_else(lhs_w, rhs_w, mask_w);
   cudf::test::expect_columns_equal(out->view(), expected_w);
}


template <typename T>
struct CopyTestTimestamp: public cudf::test::BaseFixture {};
TYPED_TEST_CASE(CopyTestTimestamp, cudf::test::TimestampTypes);

TYPED_TEST(CopyTestTimestamp, CopyIfElseTestScalarColumn) 
{ 
   using T = TypeParam;   
   
   int num_els = 4;

   bool mask[]    = { 1, 0, 0, 1 };
   bool_wrapper mask_w(mask, mask + num_els);
   
   cudf::timestamp_scalar<T> lhs_w(5);

   T rhs[]        = { 6, 6, 6, 6 };
   bool rhs_v[]   = { 1, 0, 1, 1 };
   wrapper<T> rhs_w(rhs, rhs + num_els, rhs_v);
   
   T expected[]   = { 5, 6, 6, 5 };   
   wrapper<T> expected_w(expected, expected + num_els, rhs_v);

   auto out = cudf::experimental::copy_if_else(lhs_w, rhs_w, mask_w);
   cudf::test::expect_columns_equal(out->view(), expected_w);
}

TYPED_TEST(CopyTestTimestamp, CopyIfElseTestColumnScalar) 
{ 
   using T = TypeParam;
   
   int num_els = 4;

   bool mask[]    = { 1, 0, 0, 1 };
   bool_wrapper mask_w(mask, mask + num_els);
   
   T lhs[]        = { 5, 5, 5, 5 };
   bool lhs_v[]   = { 0, 1, 1, 1 };
   wrapper<T> lhs_w(lhs, lhs + num_els, lhs_v);

   cudf::timestamp_scalar<T> rhs_w(6);
   
   T expected[]   = { 5, 6, 6, 5 };   
   wrapper<T> expected_w(expected, expected + num_els, lhs_v);

   auto out = cudf::experimental::copy_if_else(lhs_w, rhs_w, mask_w);
   cudf::test::expect_columns_equal(out->view(), expected_w);
}

TYPED_TEST(CopyTestTimestamp, CopyIfElseTestScalarScalar) 
{ 
   using T = TypeParam;
   
   int num_els = 4;   

   bool mask[]    = { 1, 0, 0, 1 };
   bool_wrapper mask_w(mask, mask + num_els);

   cudf::timestamp_scalar<T> lhs_w(5);

   cudf::timestamp_scalar<T> rhs_w(6, false);
   
   T expected[]   = { 5, 6, 6, 5 };   
   wrapper<T> expected_w(expected, expected + num_els, mask);

   auto out = cudf::experimental::copy_if_else(lhs_w, rhs_w, mask_w);
   cudf::test::expect_columns_equal(out->view(), expected_w);
}

struct CopyTestUntyped : public cudf::test::BaseFixture {};

TEST_F(CopyTestUntyped, CopyIfElseTypeMismatch)
{               
   int num_els = 4;

   bool mask[]    = { 1, 1, 1, 1 };
   bool_wrapper mask_w(mask, mask + num_els);
   
   float lhs[]    = { 5, 5, 5, 5 };   
   wrapper<float> lhs_w(lhs, lhs + num_els, mask);

   int rhs[]      = { 6, 6, 6, 6 };                      
   wrapper<int> rhs_w(rhs, rhs + num_els, mask);   

   EXPECT_THROW(  cudf::experimental::copy_if_else(lhs_w, rhs_w, mask_w),
                  cudf::logic_error);
}

struct StringsCopyIfElseTest : public cudf::test::BaseFixture {};

TEST_F(StringsCopyIfElseTest, CopyIfElse)
{
   auto valids = cudf::test::make_counting_transform_iterator(0, [](auto i) { return i % 2 == 0 ? true : false; });

   std::vector<const char*> h_strings1{ "eee", "bb", "", "aa", "bbb", "ééé" };
   cudf::test::strings_column_wrapper strings1( h_strings1.begin(), h_strings1.end(), valids );   
   std::vector<const char*> h_strings2{ "zz",  "", "yyy", "w", "ééé", "ooo" };
   cudf::test::strings_column_wrapper strings2( h_strings2.begin(), h_strings2.end(), valids );   

   bool mask[] = { 1, 1, 0, 1, 0, 1 };
   bool_wrapper mask_w(mask, mask + 6);
   
   auto results = cudf::experimental::copy_if_else(strings1, strings2, mask_w);
      
   std::vector<const char*> h_expected;
   for( cudf::size_type idx=0; idx < static_cast<cudf::size_type>(h_strings1.size()); ++idx )
   {
       if( mask[idx] )
           h_expected.push_back( h_strings1[idx] );
       else
           h_expected.push_back( h_strings2[idx] );
   }
   cudf::test::strings_column_wrapper expected( h_expected.begin(), h_expected.end(), valids);
   cudf::test::expect_columns_equal(*results,expected);
}

TEST_F(StringsCopyIfElseTest, CopyIfElseScalarColumn)
{   
   auto valids = cudf::test::make_counting_transform_iterator(0, [](auto i) { return i % 2 == 0 ? true : false; });

   std::vector<const char*> h_string1{ "eee" };   
   cudf::string_scalar strings1{h_string1[0]};
   std::vector<const char*> h_strings2{ "zz",  "", "yyy", "w", "ééé", "ooo" };
   cudf::test::strings_column_wrapper strings2( h_strings2.begin(), h_strings2.end(), valids );   

   bool mask[] = { 1, 0, 1, 0, 1, 0 };
   bool_wrapper mask_w(mask, mask + 6);  
      
   auto results = cudf::experimental::copy_if_else(strings1, strings2, mask_w);
      
   std::vector<const char*> h_expected;
   for( cudf::size_type idx=0; idx < static_cast<cudf::size_type>(h_strings2.size()); ++idx )
   {
      if( mask[idx] ){
         h_expected.push_back( h_string1[0] );
      } else {
         h_expected.push_back( h_strings2[idx] );
      }
   }
   cudf::test::strings_column_wrapper expected( h_expected.begin(), h_expected.end(), valids);      
   cudf::test::expect_columns_equal(*results,expected);
}


TEST_F(StringsCopyIfElseTest, CopyIfElseColumnScalar)
{   
   auto valids = cudf::test::make_counting_transform_iterator(0, [](auto i) { return i % 2 == 0 ? true : false; });

   std::vector<const char*> h_string1{ "eee" };
   cudf::string_scalar strings1{h_string1[0]};
   std::vector<const char*> h_strings2{ "zz",  "", "yyy", "w", "ééé", "ooo" };
   cudf::test::strings_column_wrapper strings2( h_strings2.begin(), h_strings2.end(), valids );   

   bool mask[] = { 0, 1, 1, 1, 0, 1 };
   bool_wrapper mask_w(mask, mask + 6);  
      
   auto results = cudf::experimental::copy_if_else(strings2, strings1, mask_w);
      
   std::vector<const char*> h_expected;
   for( cudf::size_type idx=0; idx < static_cast<cudf::size_type>(h_strings2.size()); ++idx )
   {
      if( mask[idx] ){
         h_expected.push_back( h_strings2[idx] );
      } else {
         h_expected.push_back( h_string1[0] );
      }
   }
   cudf::test::strings_column_wrapper expected( h_expected.begin(), h_expected.end(), valids);
   cudf::test::expect_columns_equal(*results,expected);
}

TEST_F(StringsCopyIfElseTest, CopyIfElseScalarScalar)
{   
   auto valids = cudf::test::make_counting_transform_iterator(0, [](auto i) { return i % 2 == 0 ? true : false; });

   std::vector<const char*> h_string1{ "eee" };
   cudf::string_scalar string1{h_string1[0]};
   std::vector<const char*> h_string2{ "aaa" };
   cudf::string_scalar string2{h_string2[0], false};   

   constexpr cudf::size_type mask_size = 6;   
   bool mask[] = { 1, 0, 1, 0, 1, 0 };
   bool_wrapper mask_w(mask, mask + mask_size);     
      
   auto results = cudf::experimental::copy_if_else(string1, string2, mask_w);
      
   std::vector<const char*> h_expected;
   for( cudf::size_type idx=0; idx < static_cast<cudf::size_type>(mask_size); ++idx )
   {
      if( mask[idx] ){
         h_expected.push_back( h_string1[0] );
      } else {
         h_expected.push_back( h_string2[0] );
      }
   }
   cudf::test::strings_column_wrapper expected( h_expected.begin(), h_expected.end(), valids );   
   cudf::test::expect_columns_equal(*results,expected);
}

