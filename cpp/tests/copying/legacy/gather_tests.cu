/*
 * Copyright (c) 2019, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <cudf/legacy/copying.hpp>
#include <cudf/legacy/table.hpp>

#include <gmock/gmock.h>
#include <gtest/gtest.h>
#include <tests/utilities/legacy/column_wrapper.cuh>
#include "tests/utilities/legacy/compare_column_wrappers.cuh"
#include <tests/utilities/legacy/cudf_test_fixtures.h>
#include <tests/utilities/legacy/cudf_test_utils.cuh>
#include <tests/utilities/legacy/compare_column_wrappers.cuh>
#include <cudf/types.hpp>
#include <cudf/utilities/legacy/wrapper_types.hpp>

#include <random>

template <typename T>
struct GatherTest : GdfTest {};

using test_types =
    ::testing::Types<int8_t, int16_t, int32_t, int64_t, float, double, cudf::bool8>;
TYPED_TEST_CASE(GatherTest, test_types);

TYPED_TEST(GatherTest, MultiColReverseIdentityTest) {
  constexpr cudf::size_type source_size{1000};
  constexpr cudf::size_type destination_size{1000};

  static_assert(source_size == destination_size,
                "Source and destination columns must be the same size.");

  constexpr cudf::size_type n_cols = 3;
 
  std::vector<cudf::test::column_wrapper<TypeParam>> v_src(
    n_cols,
    { source_size, 
      [](cudf::size_type row){ return static_cast<TypeParam>(row); },
      [](cudf::size_type row) { return true; }
    }
  );
  std::vector<gdf_column*> vp_src {n_cols};
  for(size_t i = 0; i < v_src.size(); i++){
    vp_src[i] = v_src[i].get();  
  }
  
  // Create gather_map that reverses order of source_column
  std::vector<cudf::size_type> host_gather_map(source_size);
  std::iota(host_gather_map.begin(), host_gather_map.end(), 0);
  std::reverse(host_gather_map.begin(), host_gather_map.end());
  thrust::device_vector<cudf::size_type> gather_map(host_gather_map);

  std::vector<cudf::test::column_wrapper<TypeParam>> v_dest(
    n_cols,
    { source_size, 
      [](cudf::size_type row){return static_cast<TypeParam>(row);},
      [](cudf::size_type row) { return true; }
    }
  );
  std::vector<gdf_column*> vp_dest {n_cols};
  for(size_t i = 0; i < v_src.size(); i++){
    vp_dest[i] = v_dest[i].get();  
  }
 
  cudf::table source_table{ vp_src };
  cudf::table destination_table{ vp_dest };

  EXPECT_NO_THROW(
      cudf::gather(&source_table, gather_map.data().get(), &destination_table));

  for(size_t i = 0; i < n_cols; i++){
    // Expected result is the reversal of the source column
    std::vector<TypeParam> expected_data;
    std::vector<cudf::valid_type> expected_bitmask;
    std::tie(expected_data, expected_bitmask) = v_src[i].to_host();
    std::reverse(expected_data.begin(), expected_data.end());

    // Copy result of destination column to host
    std::vector<TypeParam> result_data;
    std::vector<cudf::valid_type> result_bitmask;
    std::tie(result_data, result_bitmask) = v_dest[i].to_host();

    auto print_all_unequal_pairs { true };
    expect_column_values_are_equal<TypeParam>(
        destination_size, expected_data.data(), nullptr, "Expected",
        result_data.data(), nullptr, "Actual",
        print_all_unequal_pairs);

    for (cudf::size_type i = 0; i < destination_size; i++) {
      EXPECT_TRUE(gdf_is_valid(result_bitmask.data(), i))
          << "Value at index " << i << " should be non-null!\n";
    }
  }
}

TYPED_TEST(GatherTest, MultiColNullTest) {
  constexpr cudf::size_type source_size{1000};
  constexpr cudf::size_type destination_size{1000};
  
  static_assert(0 == source_size % 2,
                "Size of source data must be a multiple of 2.");
  static_assert(source_size == destination_size,
                "Source and destination columns must be equal size.");
  
  constexpr cudf::size_type n_cols = 3;
 
  // elements with even indices are null
  std::vector<cudf::test::column_wrapper<TypeParam>> v_src(
    n_cols,
    { source_size, 
      [](cudf::size_type row){ return static_cast<TypeParam>(row); },
      [](cudf::size_type row) { return row & 1; }
    }
  );
  std::vector<gdf_column*> vp_src {n_cols};
  for(size_t i = 0; i < v_src.size(); i++){
    vp_src[i] = v_src[i].get();  
  }
  
  // Create gather_map that reverses order of source_column
  std::vector<cudf::size_type> host_gather_map(source_size);
  for (cudf::size_type i = 0; i < destination_size / 2; ++i) {
    host_gather_map[i] = i * 2 + 1;
    host_gather_map[destination_size / 2 + i] = i * 2;
  }
  thrust::device_vector<cudf::size_type> gather_map(host_gather_map);

  std::vector<cudf::test::column_wrapper<TypeParam>> v_dest(
    n_cols,
    { source_size, 
      [](cudf::size_type row){return static_cast<TypeParam>(row);},
      [](cudf::size_type row) { return true; }
    }
  );
  std::vector<gdf_column*> vp_dest {n_cols};
  for(size_t i = 0; i < v_src.size(); i++){
    vp_dest[i] = v_dest[i].get();  
  }
 
  cudf::table source_table{ vp_src };
  cudf::table destination_table{ vp_dest };

  EXPECT_NO_THROW(
      cudf::gather(&source_table, gather_map.data().get(), &destination_table));

  for(size_t c = 0; c < n_cols; c++){
    // Copy result of destination column to host
    std::vector<TypeParam> result_data;
    std::vector<cudf::valid_type> result_bitmask;
    std::tie(result_data, result_bitmask) = v_dest[c].to_host();
    
    EXPECT_EQ(v_dest[c].null_count(), destination_size/2) 
      << "Null count should be " << destination_size/2 << "\n";

    for (cudf::size_type i = 0; i < destination_size; i++) {
      // The first half of the destination column should be all valid
      // and values should be 1, 3, 5, 7, etc.
      if (i < destination_size / 2) {
        EXPECT_TRUE(gdf_is_valid(result_bitmask.data(), i))
            << "Value at index " << i << " should be non-null!\n";
        EXPECT_EQ(static_cast<TypeParam>(i * 2 + 1), result_data[i]);
      } else {
        // The last half of the destination column should be all null
        EXPECT_FALSE(gdf_is_valid(result_bitmask.data(), i))
            << "Value at index " << i << " should be null!\n";
      }
    }
  }
}

TYPED_TEST(GatherTest, MultiColInPlaceTest) {
  constexpr cudf::size_type source_size{1000};
  
  static_assert(0 == source_size % 2,
                "Size of source data must be a multiple of 2.");
  
  constexpr cudf::size_type n_cols = 3;
 
  // elements with even indices are null
  std::vector<cudf::test::column_wrapper<TypeParam>> v_src(
    n_cols,
    { source_size, 
      [](cudf::size_type row){ return static_cast<TypeParam>(row); },
      [](cudf::size_type row) { return row & 1; }
    }
  );
  std::vector<gdf_column*> vp_src {n_cols};
  for(size_t i = 0; i < v_src.size(); i++){
    vp_src[i] = v_src[i].get();  
  }
  
  // Create gather_map that reverses order of source_column
  std::vector<cudf::size_type> host_gather_map(source_size);
  for (cudf::size_type i = 0; i < source_size / 2; ++i) {
    host_gather_map[i] = i * 2 + 1;
    host_gather_map[source_size / 2 + i] = i * 2;
  }
  thrust::device_vector<cudf::size_type> gather_map(host_gather_map);

  cudf::table source_table{ vp_src };

  EXPECT_NO_THROW(
      cudf::gather(&source_table, gather_map.data().get(), &source_table));

  for(size_t c = 0; c < n_cols; c++){
    // Copy result of source column to host
    std::vector<TypeParam> result_data;
    std::vector<cudf::valid_type> result_bitmask;
    std::tie(result_data, result_bitmask) = v_src[c].to_host();
    
    EXPECT_EQ(v_src[c].null_count(), source_size/2) 
      << "Null count should be " << source_size/2 << "\n";

    for (cudf::size_type i = 0; i < source_size; i++) {
      // The first half of the source column should be all valid
      // and values should be 1, 3, 5, 7, etc.
      if (i < source_size / 2) {
        EXPECT_TRUE(gdf_is_valid(result_bitmask.data(), i))
            << "Value at index " << i << " should be non-null!\n";
        EXPECT_EQ(static_cast<TypeParam>(i * 2 + 1), result_data[i]);
      } else {
        // The last half of the source column should be all null
        EXPECT_FALSE(gdf_is_valid(result_bitmask.data(), i))
            << "Value at index " << i << " should be null!\n";
      }
    }
  }
}

TYPED_TEST(GatherTest, DtypeMistach){
  constexpr cudf::size_type source_size{1000};
  constexpr cudf::size_type destination_size{1000};

  cudf::test::column_wrapper<int32_t> source{source_size};
  cudf::test::column_wrapper<float> destination{destination_size};

  gdf_column * raw_source = source.get();
  gdf_column * raw_destination = destination.get();

  cudf::table source_table{&raw_source, 1};
  cudf::table destination_table{&raw_destination, 1};

  rmm::device_vector<cudf::size_type> gather_map(source_size);

  EXPECT_THROW(cudf::gather(&source_table, gather_map.data().get(),
                             &destination_table), cudf::logic_error);
}

TYPED_TEST(GatherTest, DestMissingValid){
  constexpr cudf::size_type source_size{1000};
  constexpr cudf::size_type destination_size{1000};

  cudf::test::column_wrapper<TypeParam> source(source_size, true);
  cudf::test::column_wrapper<TypeParam> destination(destination_size, false);

  gdf_column * raw_source = source.get();
  gdf_column * raw_destination = destination.get();

  cudf::table source_table{&raw_source, 1};
  cudf::table destination_table{&raw_destination, 1};

  rmm::device_vector<cudf::size_type> gather_map(source_size);

  EXPECT_THROW(cudf::gather(&source_table, gather_map.data().get(),
                             &destination_table), cudf::logic_error);
}

TYPED_TEST(GatherTest, NumColumnsMismatch){
  constexpr cudf::size_type source_size{1000};
  constexpr cudf::size_type destination_size{1000};

  cudf::test::column_wrapper<TypeParam> source0(source_size, true);
  cudf::test::column_wrapper<TypeParam> source1(source_size, true);
  cudf::test::column_wrapper<TypeParam> destination(destination_size, false);

  std::vector<gdf_column*> source_cols{source0.get(), source1.get()};

  gdf_column * raw_destination = destination.get();

  cudf::table source_table{source_cols.data(), 2};
  cudf::table destination_table{&raw_destination, 1};

  rmm::device_vector<cudf::size_type> gather_map(source_size);

  EXPECT_THROW(cudf::gather(&source_table, gather_map.data().get(),
                             &destination_table), cudf::logic_error);
}

TYPED_TEST(GatherTest, IdentityTest) {
  constexpr cudf::size_type source_size{1000};
  constexpr cudf::size_type destination_size{1000};

  cudf::test::column_wrapper<TypeParam> source_column{
      source_size, [](cudf::size_type row) { return static_cast<TypeParam>(row); },
      [](cudf::size_type row) { return true; }};

  thrust::device_vector<cudf::size_type> gather_map(destination_size);
  thrust::sequence(gather_map.begin(), gather_map.end());

  cudf::test::column_wrapper<TypeParam> destination_column(destination_size,
                                                           true);

  gdf_column* raw_source = source_column.get();
  gdf_column* raw_destination = destination_column.get();

  cudf::table source_table{&raw_source, 1};
  cudf::table destination_table{&raw_destination, 1};

  EXPECT_NO_THROW(
      cudf::gather(&source_table, gather_map.data().get(), &destination_table));

  expect_columns_are_equal<TypeParam>(source_column, destination_column);
}

TYPED_TEST(GatherTest, ReverseIdentityTest) {
  constexpr cudf::size_type source_size{1000};
  constexpr cudf::size_type destination_size{1000};

  static_assert(source_size == destination_size,
                "Source and destination columns must be the same size.");

  cudf::test::column_wrapper<TypeParam> source_column{
      source_size, [](cudf::size_type row) { return static_cast<TypeParam>(row); },
      [](cudf::size_type row) { return true; }};

  // Create gather_map that reverses order of source_column
  std::vector<cudf::size_type> host_gather_map(source_size);
  std::iota(host_gather_map.begin(), host_gather_map.end(), 0);
  std::reverse(host_gather_map.begin(), host_gather_map.end());
  thrust::device_vector<cudf::size_type> gather_map(host_gather_map);

  cudf::test::column_wrapper<TypeParam> destination_column(destination_size,
                                                           true);

  gdf_column* raw_source = source_column.get();
  gdf_column* raw_destination = destination_column.get();

  cudf::table source_table{&raw_source, 1};
  cudf::table destination_table{&raw_destination, 1};

  EXPECT_NO_THROW(
      cudf::gather(&source_table, gather_map.data().get(), &destination_table));

  // Expected result is the reversal of the source column
  std::vector<TypeParam> expected_data;
  std::vector<cudf::valid_type> expected_bitmask;
  std::tie(expected_data, expected_bitmask) = source_column.to_host();
  std::reverse(expected_data.begin(), expected_data.end());

  // Copy result of destination column to host
  std::vector<TypeParam> result_data;
  std::vector<cudf::valid_type> result_bitmask;
  std::tie(result_data, result_bitmask) = destination_column.to_host();

  auto print_all_unequal_pairs { true };
  expect_column_values_are_equal<TypeParam>(
      destination_size, expected_data.data(), nullptr, "Expected",
      result_data.data(), nullptr, "Actual",
      print_all_unequal_pairs);

  for (cudf::size_type i = 0; i < destination_size; i++) {
    EXPECT_TRUE(gdf_is_valid(result_bitmask.data(), i))
        << "Value at index " << i << " should be non-null!\n";
  }
}

TYPED_TEST(GatherTest, AllNull) {
  constexpr cudf::size_type source_size{1000};
  constexpr cudf::size_type destination_size{1000};

  // source column has all null values
  cudf::test::column_wrapper<TypeParam> source_column{
      source_size, [](cudf::size_type row) { return static_cast<TypeParam>(row); },
      [](cudf::size_type row) { return false; }};

  // Create gather_map that gathers to random locations
  std::vector<cudf::size_type> host_gather_map(source_size);
  std::iota(host_gather_map.begin(), host_gather_map.end(), 0);
  std::mt19937 g(0);
  std::shuffle(host_gather_map.begin(), host_gather_map.end(), g);
  thrust::device_vector<cudf::size_type> gather_map(host_gather_map);

  cudf::test::column_wrapper<TypeParam> destination_column(destination_size,
                                                           true);

  gdf_column* raw_source = source_column.get();
  gdf_column* raw_destination = destination_column.get();

  cudf::table source_table{&raw_source, 1};
  cudf::table destination_table{&raw_destination, 1};

  EXPECT_NO_THROW(
      cudf::gather(&source_table, gather_map.data().get(), &destination_table));

  // Copy result of destination column to host
  std::vector<TypeParam> result_data;
  std::vector<cudf::valid_type> result_bitmask;
  std::tie(result_data, result_bitmask) = destination_column.to_host();

  // All values of result should be null
  for (cudf::size_type i = 0; i < destination_size; i++) {
    EXPECT_FALSE(gdf_is_valid(result_bitmask.data(), i))
        << "Value at index " << i << " should be null!\n";
  }
}

TYPED_TEST(GatherTest, EveryOtherNull) {
  constexpr cudf::size_type source_size{1234};
  constexpr cudf::size_type destination_size{source_size};

  static_assert(0 == source_size % 2,
                "Size of source data must be a multiple of 2.");
  static_assert(source_size == destination_size,
                "Source and destination columns must be equal size.");

  // elements with even indices are null
  cudf::test::column_wrapper<TypeParam> source_column{
      source_size, [](cudf::size_type row) { return static_cast<TypeParam>(row); },
      [](cudf::size_type row) { return row % 2; }};

  // Gather null values to the last half of the destination column
  std::vector<cudf::size_type> host_gather_map(source_size);
  for (cudf::size_type i = 0; i < destination_size / 2; ++i) {
    host_gather_map[i] = i * 2 + 1;
    host_gather_map[destination_size / 2 + i] = i * 2;
  }

  thrust::device_vector<cudf::size_type> gather_map(host_gather_map);

  cudf::test::column_wrapper<TypeParam> destination_column(destination_size,
                                                           true);

  gdf_column* raw_source = source_column.get();
  gdf_column* raw_destination = destination_column.get();

  cudf::table source_table{&raw_source, 1};
  cudf::table destination_table{&raw_destination, 1};

  EXPECT_NO_THROW(
      cudf::gather(&source_table, gather_map.data().get(), &destination_table));

  // Copy result of destination column to host
  std::vector<TypeParam> result_data;
  std::vector<cudf::valid_type> result_bitmask;
  std::tie(result_data, result_bitmask) = destination_column.to_host();

  for (cudf::size_type i = 0; i < destination_size; i++) {
    // The first half of the destination column should be all valid
    // and values should be 1, 3, 5, 7, etc.
    if (i < destination_size / 2) {
      EXPECT_TRUE(gdf_is_valid(result_bitmask.data(), i))
          << "Value at index " << i << " should be non-null!\n";
      EXPECT_EQ(static_cast<TypeParam>(i * 2 + 1), result_data[i]);
    } else {
      // The last half of the destination column should be all null
      EXPECT_FALSE(gdf_is_valid(result_bitmask.data(), i))
          << "Value at index " << i << " should be null!\n";
    }
  }
}
