#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019-2020, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <functional>
#include <limits>
#include <memory>
#include <type_traits>
#include <tests/utilities/base_fixture.hpp>
#include <tests/utilities/column_utilities.hpp>
#include <tests/utilities/column_wrapper.hpp>
#include <tests/utilities/cudf_gtest.hpp>
#include <tests/utilities/type_lists.hpp>
#include <cudf/column/column.hpp>
#include <cudf/copying.hpp>
#include <cudf/scalar/scalar.hpp>

using cudf::test::fixed_width_column_wrapper;
using TestTypes = cudf::test::Types<int32_t>;

template<typename T,
         typename ScalarType = cudf::experimental::scalar_type_t<T>>
std::unique_ptr<cudf::scalar>
make_scalar(
    hipStream_t stream = 0,
    rmm::mr::device_memory_resource* mr = rmm::mr::get_default_resource()) {
      auto s = new ScalarType(0, false, stream, mr);
      return std::unique_ptr<cudf::scalar>(s);
}

template<typename T,
         typename ScalarType = cudf::experimental::scalar_type_t<T>>
std::unique_ptr<cudf::scalar>
make_scalar(
    T value,
    hipStream_t stream = 0,
    rmm::mr::device_memory_resource* mr = rmm::mr::get_default_resource()) {
      auto s = new ScalarType(value, true, stream, mr);
      return std::unique_ptr<cudf::scalar>(s);
}

template<typename T>
auto lowest = std::numeric_limits<T>::lowest();

template<typename T>
auto highest = std::numeric_limits<T>::max();

template <typename T>
struct ShiftTest : public cudf::test::BaseFixture {};

TYPED_TEST_CASE(ShiftTest, cudf::test::FixedWidthTypes);

TYPED_TEST(ShiftTest, OneColumnEmpty)
{
    using T =  TypeParam;

    auto input = fixed_width_column_wrapper<T> {};
    auto expected = fixed_width_column_wrapper<T> ({}, {});

    auto fill = make_scalar<T>();
    auto actual = cudf::experimental::shift(input, 5, *fill);

    cudf::test::expect_columns_equal(expected, *actual);
}

TYPED_TEST(ShiftTest, TwoColumnsEmpty)
{
    using T =  TypeParam;

    auto input = fixed_width_column_wrapper<T> ({}, {});
    auto expected = fixed_width_column_wrapper<T> ({}, {});

    auto fill = make_scalar<T>();
    auto actual = cudf::experimental::shift(input, 5, *fill);

    cudf::test::expect_columns_equal(expected, *actual);
}

TYPED_TEST(ShiftTest, OneColumn)
{
    using T =  TypeParam;

    auto input = fixed_width_column_wrapper<T>{ lowest<T>, 1, 2, 3, 4, 5, highest<T> };
    auto expected = fixed_width_column_wrapper<T>{ 7, 7, lowest<T>, 1, 2, 3, 4 };

    auto fill = make_scalar<T>(7);
    auto actual = cudf::experimental::shift(input, 2, *fill);

    cudf::test::expect_columns_equal(expected, *actual);
}

TYPED_TEST(ShiftTest, OneColumnNegativeShift)
{
    using T =  TypeParam;

    auto input = fixed_width_column_wrapper<T>{ lowest<T>, 1, 2, 3, 4, 5, highest<T> };
    auto expected = fixed_width_column_wrapper<T>{ 4, 5, highest<T>, 7, 7, 7, 7 };

    auto fill = make_scalar<T>(7);
    auto actual = cudf::experimental::shift(input, -4, *fill);

    cudf::test::expect_columns_equal(expected, *actual);
}

TYPED_TEST(ShiftTest, OneColumnNullFill)
{
    using T =  TypeParam;

    auto input = fixed_width_column_wrapper<T>{ lowest<T>, 5, 0, 3, 0, 1, highest<T> };
    auto expected = fixed_width_column_wrapper<T>({ 0, 0, lowest<T>, 5, 0, 3, 0 }, { 0, 0, 1, 1, 1, 1, 1 });

    auto fill = make_scalar<T>();

    auto actual = cudf::experimental::shift(input, 2, *fill);

    cudf::test::expect_columns_equal(expected, *actual);
}

TYPED_TEST(ShiftTest, TwoColumnsNullableInput)
{
    using T =  TypeParam;

    auto input = fixed_width_column_wrapper<T>({ 1, 2, 3, 4, 5 }, { 0, 1, 1, 1, 0});
    auto expected = fixed_width_column_wrapper<T>({ 7, 7, 1, 2, 3 }, { 1, 1, 0, 1, 1});

    auto fill = make_scalar<T>(7);
    auto actual = cudf::experimental::shift(input, 2, *fill);

    cudf::test::expect_columns_equal(expected, *actual);
}

TYPED_TEST(ShiftTest, MismatchFillValueDtypes)
{
    using T = TypeParam;

    if (std::is_same<T, int>::value) {
        return;
    }

    auto input = fixed_width_column_wrapper<T> {};

    auto fill = make_scalar<int>();

    std::unique_ptr<cudf::column> output;

    EXPECT_THROW(output = cudf::experimental::shift(input, 5, *fill),
                 cudf::logic_error);
}
