/*
 * Copyright (c) 2020, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */
#include <tests/strings/utilities.h>
#include <cudf/column/column_view.hpp>
#include <cudf/copying.hpp>
#include <cudf/detail/gather.cuh>
#include <cudf/detail/gather.hpp>
#include <cudf/table/table.hpp>
#include <cudf/table/table_view.hpp>
#include <tests/utilities/base_fixture.hpp>
#include <tests/utilities/column_utilities.hpp>
#include <tests/utilities/column_wrapper.hpp>
#include <tests/utilities/cudf_gtest.hpp>
#include <tests/utilities/table_utilities.hpp>
#include <tests/utilities/type_lists.hpp>

template <typename T>
class GatherTestList : public cudf::test::BaseFixture {
};
using FixedWidthTypesNotBool = cudf::test::Concat<cudf::test::IntegralTypesNotBool,
                                                  cudf::test::FloatingPointTypes,
                                                  cudf::test::TimestampTypes>;
TYPED_TEST_CASE(GatherTestList, FixedWidthTypesNotBool);

TYPED_TEST(GatherTestList, Gather)
{
  using T = TypeParam;

  // List<T>
  cudf::test::lists_column_wrapper<T> list{{1, 2, 3, 4}, {5}, {6, 7}, {8, 9, 10}};
  cudf::test::fixed_width_column_wrapper<int> gather_map{0, 2};

  cudf::table_view source_table({list});
  auto results = cudf::gather(source_table, gather_map);

  cudf::test::lists_column_wrapper<T> expected{{1, 2, 3, 4}, {6, 7}};

  cudf::test::expect_columns_equal(results->view().column(0), expected);
}

TYPED_TEST(GatherTestList, GatherNulls)
{
  using T = TypeParam;

  auto valids = cudf::test::make_counting_transform_iterator(
    0, [](auto i) { return i % 2 == 0 ? true : false; });

  // List<T>
  cudf::test::lists_column_wrapper<T> list{
    {{1, 2, 3, 4}, valids}, {5}, {{6, 7}, valids}, {{8, 9, 10}, valids}};
  cudf::test::fixed_width_column_wrapper<int> gather_map{0, 2};

  cudf::table_view source_table({list});
  auto results = cudf::gather(source_table, gather_map);

  cudf::test::lists_column_wrapper<T> expected{{{1, 2, 3, 4}, valids}, {{6, 7}, valids}};

  cudf::test::expect_columns_equal(results->view().column(0), expected);
}

TYPED_TEST(GatherTestList, GatherNested)
{
  using T   = TypeParam;
  using LCW = cudf::test::lists_column_wrapper<T>;

  // List<List<T>>
  {
    cudf::test::lists_column_wrapper<T> list{{{2, 3}, {4, 5}},
                                             {{6, 7, 8}, {9, 10, 11}, {12, 13, 14}},
                                             {{15, 16}, {17, 18}, {17, 18}, {17, 18}, {17, 18}}};
    cudf::test::fixed_width_column_wrapper<int> gather_map{0, 2};

    cudf::table_view source_table({list});
    auto results = cudf::gather(source_table, gather_map);

    cudf::test::lists_column_wrapper<T> expected{
      {{2, 3}, {4, 5}}, {{15, 16}, {17, 18}, {17, 18}, {17, 18}, {17, 18}}};

    cudf::test::expect_columns_equal(results->view().column(0), expected);
  }

  // List<List<List<T>>>
  {
    cudf::test::lists_column_wrapper<T> list{
      {{{2, 3}, {4, 5}}, {{6, 7, 8}, {9, 10, 11}, {12, 13, 14}}},
      {{{15, 16}, {17, 18}, {17, 18}, {17, 18}, {17, 18}}},
      {{LCW{0}}},
      {{{10}, {20, 30, 40, 50}, {60, 70, 80}},
       {{0, 1, 3}, {5}},
       {{11, 12, 13, 14, 15}, {16, 17}, {0}}},
      {{{10, 20}}, {LCW{30}}, {{40, 50}, {60, 70, 80}}}};
    cudf::test::fixed_width_column_wrapper<int> gather_map{1, 2, 4};

    cudf::table_view source_table({list});
    auto results = cudf::gather(source_table, gather_map);

    cudf::test::lists_column_wrapper<T> expected{
      {{{15, 16}, {17, 18}, {17, 18}, {17, 18}, {17, 18}}},
      {{LCW{0}}},
      {{{10, 20}}, {LCW{30}}, {{40, 50}, {60, 70, 80}}}};

    cudf::test::expect_columns_equal(results->view().column(0), expected);
  }
}

TYPED_TEST(GatherTestList, GatherNestedForceRecycle)
{
  using T   = TypeParam;
  using LCW = cudf::test::lists_column_wrapper<T>;

  // these cases force the temporary memory-recycling behavior internal
  // to the gather() recursion

  // recycled on both levels
  // List<List<List<T>>>
  {
    cudf::test::lists_column_wrapper<T> list{
      {{LCW{2}}}, {{LCW{3}}}, {{LCW{5}}}, {{LCW{6}}}, {{LCW{7}}}};

    cudf::test::fixed_width_column_wrapper<int> gather_map{0, 1, 2};

    cudf::table_view source_table({list});
    auto results = cudf::gather(source_table, gather_map);

    cudf::test::lists_column_wrapper<T> expected{{{LCW{2}}}, {{LCW{3}}}, {{LCW{5}}}};

    cudf::test::expect_columns_equal(results->view().column(0), expected);
  }

  // recycled on first level but not second
  // List<List<List<T>>>
  {
    cudf::test::lists_column_wrapper<T> list{
      {{LCW{2}}}, {{LCW{3}, LCW{4}}}, {{LCW{5}}}, {{LCW{6}}}, {{LCW{7}}}};

    cudf::test::fixed_width_column_wrapper<int> gather_map{0, 1, 2};

    cudf::table_view source_table({list});
    auto results = cudf::gather(source_table, gather_map);

    cudf::test::lists_column_wrapper<T> expected{{{LCW{2}}}, {{LCW{3}, LCW{4}}}, {{LCW{5}}}};

    cudf::test::expect_columns_equal(results->view().column(0), expected);
  }

  // recycled on both levels
  // List<List<List<T>>>
  {
    cudf::test::lists_column_wrapper<T> list{
      {{LCW{2}}}, {{LCW{}}}, {{LCW{5}}}, {{LCW{6}}}, {{LCW{7}}}};

    cudf::test::fixed_width_column_wrapper<int> gather_map{0, 1, 2};

    cudf::table_view source_table({list});
    auto results = cudf::gather(source_table, gather_map);

    cudf::test::lists_column_wrapper<T> expected{{{LCW{2}}}, {{LCW{}}}, {{LCW{5}}}};

    cudf::test::expect_columns_equal(results->view().column(0), expected);
  }
}

TYPED_TEST(GatherTestList, GatherOutOfOrder)
{
  using T   = TypeParam;
  using LCW = cudf::test::lists_column_wrapper<T>;

  // List<List<T>>
  {
    cudf::test::lists_column_wrapper<T> list{{{2, 3}, {4, 5}},
                                             {{6, 7, 8}, {9, 10, 11}, {12, 13, 14}},
                                             {{15, 16}, {17, 18}, {17, 18}, {17, 18}, {17, 18}}};
    cudf::test::fixed_width_column_wrapper<int> gather_map{1, 2, 0};

    cudf::table_view source_table({list});
    auto results = cudf::gather(source_table, gather_map);

    cudf::test::lists_column_wrapper<T> expected{{{6, 7, 8}, {9, 10, 11}, {12, 13, 14}},
                                                 {{15, 16}, {17, 18}, {17, 18}, {17, 18}, {17, 18}},
                                                 {{2, 3}, {4, 5}}};

    cudf::test::expect_columns_equal(results->view().column(0), expected);
  }
}

TYPED_TEST(GatherTestList, GatherNestedNulls)
{
  using T   = TypeParam;
  using LCW = cudf::test::lists_column_wrapper<T>;

  auto valids = cudf::test::make_counting_transform_iterator(
    0, [](auto i) { return i % 2 == 0 ? true : false; });

  // List<List<T>>
  {
    cudf::test::lists_column_wrapper<T> list{
      {{{2, 3}, valids}, {4, 5}},
      {{{6, 7, 8}, {9, 10, 11}, {12, 13, 14}}, valids},
      {{15, 16}, {17, 18}, {17, 18}, {17, 18}, {17, 18}},
      {{{{25, 26}, valids}, {27, 28}, {{29, 30}, valids}, {31, 32}, {33, 34}}, valids}};

    cudf::test::fixed_width_column_wrapper<int> gather_map{0, 1, 3};

    cudf::table_view source_table({list});
    auto results = cudf::gather(source_table, gather_map);

    cudf::test::lists_column_wrapper<T> expected{
      {{{2, 3}, valids}, {4, 5}},
      {{{6, 7, 8}, {9, 10, 11}, {12, 13, 14}}, valids},
      {{{{25, 26}, valids}, {27, 28}, {{29, 30}, valids}, {31, 32}, {33, 34}}, valids}};

    cudf::test::expect_columns_equal(results->view().column(0), expected);
  }

  // List<List<List<T>>>
  {
    cudf::test::lists_column_wrapper<T> list{
      {{{2, 3}, {4, 5}}, {{6, 7, 8}, {9, 10, 11}, {12, 13, 14}}},
      {{{15, 16}, {{27, 28}, valids}, {{37, 38}, valids}, {47, 48}, {57, 58}}},
      {{LCW{0}}},
      {{{10}, {20, 30, 40, 50}, {60, 70, 80}},
       {{0, 1, 3}, {5}},
       {{11, 12, 13, 14, 15}, {16, 17}, {0}}},
      {{{{{10, 20}, valids}}, {LCW{30}}, {{40, 50}, {60, 70, 80}}}, valids}};

    cudf::test::fixed_width_column_wrapper<int> gather_map{1, 2, 4};

    cudf::table_view source_table({list});
    auto results = cudf::gather(source_table, gather_map);

    cudf::test::lists_column_wrapper<T> expected{
      {{{15, 16}, {{27, 28}, valids}, {{37, 38}, valids}, {47, 48}, {57, 58}}},
      {{LCW{0}}},
      {{{{{10, 20}, valids}}, {LCW{30}}, {{40, 50}, {60, 70, 80}}}, valids}};

    cudf::test::expect_columns_equal(results->view().column(0), expected);
  }
}

TYPED_TEST(GatherTestList, GatherNestedWithEmpties)
{
  using T   = TypeParam;
  using LCW = cudf::test::lists_column_wrapper<T>;

  cudf::test::lists_column_wrapper<T> list{
    {{2, 3}, LCW{}}, {{6, 7, 8}, {9, 10, 11}, {12, 13, 14}}, {LCW{}}};
  cudf::test::fixed_width_column_wrapper<int> gather_map{0, 2};

  cudf::table_view source_table({list});
  auto results = cudf::gather(source_table, gather_map);

  cudf::test::lists_column_wrapper<T> expected{{{2, 3}, LCW{}}, {LCW{}}};

  cudf::test::expect_columns_equal(results->view().column(0), expected);
}

TYPED_TEST(GatherTestList, GatherDetailInvalidIndex)
{
  using T   = TypeParam;
  using LCW = cudf::test::lists_column_wrapper<T>;

  // List<List<T>>
  {
    cudf::test::lists_column_wrapper<T> list{{{2, 3}, {4, 5}},
                                             {{6, 7, 8}, {9, 10, 11}, {12, 13, 14}},
                                             {{15, 16}, {17, 18}, {17, 18}, {17, 18}, {17, 18}}};
    cudf::test::fixed_width_column_wrapper<int> gather_map{0, 15, 16, 2};

    cudf::table_view source_table({list});
    auto results = cudf::detail::gather(source_table,
                                        gather_map,
                                        cudf::detail::out_of_bounds_policy::IGNORE,
                                        cudf::detail::negative_index_policy::NOT_ALLOWED);

    std::vector<int32_t> expected_validity{1, 0, 0, 1};
    cudf::test::lists_column_wrapper<T> expected{
      {{{2, 3}, {4, 5}}, {LCW{}}, {LCW{}}, {{15, 16}, {17, 18}, {17, 18}, {17, 18}, {17, 18}}},
      expected_validity.begin()};

    cudf::test::expect_columns_equal(results->view().column(0), expected);
  }
}
