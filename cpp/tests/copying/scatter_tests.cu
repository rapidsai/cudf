/*
 * Copyright (c) 2019, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <thrust/device_vector.h>
#include "copying.hpp"
#include "gmock/gmock.h"
#include "gtest/gtest.h"
#include "tests/utilities/column_wrapper.cuh"
#include "tests/utilities/cudf_test_fixtures.h"
#include "tests/utilities/cudf_test_utils.cuh"
#include "types.hpp"
#include <random>

template <typename T>
struct ScatterTest : GdfTest {};

using test_types =
    ::testing::Types<int8_t, int16_t, int32_t, int64_t, float, double>;
TYPED_TEST_CASE(ScatterTest, test_types);


TYPED_TEST(ScatterTest, DtypeMistach){
  constexpr gdf_size_type source_size{1000};
  constexpr gdf_size_type destination_size{1000};

  cudf::test::column_wrapper<int32_t> source{source_size};
  cudf::test::column_wrapper<float> destination{destination_size};

  gdf_column * raw_source = source.get();
  gdf_column * raw_destination = destination.get();

  cudf::table source_table{&raw_source, 1};
  cudf::table destination_table{&raw_destination, 1};

  rmm::device_vector<gdf_index_type> scatter_map(source_size);

  EXPECT_THROW(cudf::scatter(&source_table, scatter_map.data().get(),
                             &destination_table), cudf::logic_error);
}

TYPED_TEST(ScatterTest, DestMissingValid){
  constexpr gdf_size_type source_size{1000};
  constexpr gdf_size_type destination_size{1000};

  cudf::test::column_wrapper<TypeParam> source{source_size, true};
  cudf::test::column_wrapper<TypeParam> destination{destination_size, false};

  gdf_column * raw_source = source.get();
  gdf_column * raw_destination = destination.get();

  cudf::table source_table{&raw_source, 1};
  cudf::table destination_table{&raw_destination, 1};

  rmm::device_vector<gdf_index_type> scatter_map(source_size);

  EXPECT_THROW(cudf::scatter(&source_table, scatter_map.data().get(),
                             &destination_table), cudf::logic_error);
}

TYPED_TEST(ScatterTest, NumColumnsMismatch){
  constexpr gdf_size_type source_size{1000};
  constexpr gdf_size_type destination_size{1000};

  cudf::test::column_wrapper<TypeParam> source0{source_size, true};
  cudf::test::column_wrapper<TypeParam> source1{source_size, true};
  cudf::test::column_wrapper<TypeParam> destination{destination_size, false};

  std::vector<gdf_column*> source_cols{source0.get(), source1.get()};

  gdf_column * raw_destination = destination.get();

  cudf::table source_table{source_cols.data(), 2};
  cudf::table destination_table{&raw_destination, 1};

  rmm::device_vector<gdf_index_type> scatter_map(source_size);

  EXPECT_THROW(cudf::scatter(&source_table, scatter_map.data().get(),
                             &destination_table), cudf::logic_error);
}

TYPED_TEST(ScatterTest, IdentityTest) {
  constexpr gdf_size_type source_size{1000};
  constexpr gdf_size_type destination_size{1000};

  cudf::test::column_wrapper<TypeParam> source_column{
      source_size, [](gdf_index_type row) { return row; },
      [](gdf_index_type row) { return true; }};

  thrust::device_vector<gdf_index_type> scatter_map(source_size);
  thrust::sequence(scatter_map.begin(), scatter_map.end());

  cudf::test::column_wrapper<TypeParam> destination_column(destination_size,
                                                           true);

  gdf_column* raw_source = source_column.get();
  gdf_column* raw_destination = destination_column.get();

  cudf::table source_table{&raw_source, 1};
  cudf::table destination_table{&raw_destination, 1};

  EXPECT_NO_THROW(cudf::scatter(&source_table, scatter_map.data().get(),
                                &destination_table));

  EXPECT_TRUE(source_column == destination_column);
}

TYPED_TEST(ScatterTest, ReverseIdentityTest) {
  constexpr gdf_size_type source_size{1000};
  constexpr gdf_size_type destination_size{1000};

  cudf::test::column_wrapper<TypeParam> source_column{
      source_size, [](gdf_index_type row) { return row; },
      [](gdf_index_type row) { return true; }};

  // Create scatter_map that reverses order of source_column
  std::vector<gdf_index_type> host_scatter_map(source_size);
  std::iota(host_scatter_map.begin(), host_scatter_map.end(), 0);
  std::reverse(host_scatter_map.begin(), host_scatter_map.end());
  thrust::device_vector<gdf_index_type> scatter_map(host_scatter_map);

  cudf::test::column_wrapper<TypeParam> destination_column(destination_size,
                                                           true);

  gdf_column* raw_source = source_column.get();
  gdf_column* raw_destination = destination_column.get();

  cudf::table source_table{&raw_source, 1};
  cudf::table destination_table{&raw_destination, 1};

  EXPECT_NO_THROW(cudf::scatter(&source_table, scatter_map.data().get(),
                                &destination_table));

  // Expected result is the reversal of the source column
  std::vector<TypeParam> expected_data;
  std::vector<gdf_valid_type> expected_bitmask;
  std::tie(expected_data, expected_bitmask) = source_column.to_host();
  std::reverse(expected_data.begin(), expected_data.end());

  // Copy result of destination column to host
  std::vector<TypeParam> result_data;
  std::vector<gdf_valid_type> result_bitmask;
  std::tie(result_data, result_bitmask) = destination_column.to_host();

  for (gdf_index_type i = 0; i < destination_size; i++) {
    EXPECT_EQ(expected_data[i], result_data[i])
        << "Data at index " << i << " doesn't match!\n";
    EXPECT_TRUE(gdf_is_valid(result_bitmask.data(), i))
        << "Value at index " << i << " should be non-null!\n";
  }
}

TYPED_TEST(ScatterTest, AllNull) {
  constexpr gdf_size_type source_size{1000};
  constexpr gdf_size_type destination_size{1000};

  // source column has all null values
  cudf::test::column_wrapper<TypeParam> source_column{
      source_size, [](gdf_index_type row) { return row; },
      [](gdf_index_type row) { return false; }};

  // Create scatter_map that scatters to random locations
  std::vector<gdf_index_type> host_scatter_map(source_size);
  std::iota(host_scatter_map.begin(), host_scatter_map.end(), 0);
  std::mt19937 g(0);
  std::shuffle(host_scatter_map.begin(), host_scatter_map.end(), g);
  thrust::device_vector<gdf_index_type> scatter_map(host_scatter_map);

  cudf::test::column_wrapper<TypeParam> destination_column(destination_size,
                                                           true);

  gdf_column* raw_source = source_column.get();
  gdf_column* raw_destination = destination_column.get();

  cudf::table source_table{&raw_source, 1};
  cudf::table destination_table{&raw_destination, 1};

  EXPECT_NO_THROW(cudf::scatter(&source_table, scatter_map.data().get(),
                                &destination_table));

  // Copy result of destination column to host
  std::vector<TypeParam> result_data;
  std::vector<gdf_valid_type> result_bitmask;
  std::tie(result_data, result_bitmask) = destination_column.to_host();

  // All values of result should be null
  for (gdf_index_type i = 0; i < destination_size; i++) {
    EXPECT_FALSE(gdf_is_valid(result_bitmask.data(), i))
        << "Value at index " << i << " should be null!\n";
  }
}

TYPED_TEST(ScatterTest, EveryOtherNull) {
  constexpr gdf_size_type source_size{1234};
  constexpr gdf_size_type destination_size{source_size};

  static_assert(0 == source_size % 2,
                "Size of source data must be a multiple of 2.");
  static_assert(source_size == destination_size,
                "Source and destination columns must be equal size.");

  // elements with even indices are null
  cudf::test::column_wrapper<TypeParam> source_column{
      source_size, [](gdf_index_type row) { return row; },
      [](gdf_index_type row) { return row % 2; }};

  // Scatter null values to the last half of the destination column
  std::vector<gdf_index_type> host_scatter_map(source_size);
  for (gdf_size_type i = 0; i < source_size / 2; ++i) {
    host_scatter_map[i * 2] = destination_size / 2 + i;
    host_scatter_map[i * 2 + 1] = i;
  }
  thrust::device_vector<gdf_index_type> scatter_map(host_scatter_map);

  cudf::test::column_wrapper<TypeParam> destination_column(destination_size,
                                                           true);

  gdf_column* raw_source = source_column.get();
  gdf_column* raw_destination = destination_column.get();

  cudf::table source_table{&raw_source, 1};
  cudf::table destination_table{&raw_destination, 1};

  EXPECT_NO_THROW(cudf::scatter(&source_table, scatter_map.data().get(),
                                &destination_table));

  // Copy result of destination column to host
  std::vector<TypeParam> result_data;
  std::vector<gdf_valid_type> result_bitmask;
  std::tie(result_data, result_bitmask) = destination_column.to_host();

  for (gdf_index_type i = 0; i < destination_size; i++) {
    // The first half of the destination column should be all valid
    // and values should be 1, 3, 5, 7, etc.
    if (i < destination_size / 2) {
      EXPECT_TRUE(gdf_is_valid(result_bitmask.data(), i))
          << "Value at index " << i << " should be non-null!\n";
      EXPECT_EQ(static_cast<TypeParam>(1 + i * 2), result_data[i]);
    }
    // The last half of the destination column should be all null
    else {
      EXPECT_FALSE(gdf_is_valid(result_bitmask.data(), i))
          << "Value at index " << i << " should be null!\n";
    }
  }
}
