#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <cudf/detail/iterator.cuh>                // include iterator header
#include <cudf/detail/utilities/transform_unary_functions.cuh>  //for meanvar

#include <bitset>
#include <cstdint>
#include <iostream>
#include <numeric>
#include <random>

#include <tests/utilities/base_fixture.hpp>
#include <tests/utilities/cudf_gmock.hpp>
#include <gmock/gmock.h>
#include <tests/utilities/type_lists.hpp>
#include <tests/utilities/column_wrapper.hpp>

#include <thrust/equal.h>
#include <thrust/transform.h>
#include <thrust/functional.h>

// for reduction tests
#include <cub/device/device_reduce.cuh>
#include <thrust/device_vector.h>

// ---------------------------------------------------------------------------

template <typename T>
T random_int(T min, T max)
{
  static unsigned seed = 13377331;
  static std::mt19937 engine{seed};
  static std::uniform_int_distribution<T> uniform{min, max};

  return uniform(engine);
}

bool random_bool()
{
  static unsigned seed = 13377331;
  static std::mt19937 engine{seed};
  static std::uniform_int_distribution<int> uniform{0, 1};

  return static_cast<bool>(uniform(engine));
}

template <typename T>
std::ostream& operator<<(std::ostream& os, cudf::meanvar<T> const& rhs)
{
  return os << "[" << rhs.value <<
               ", " << rhs.value_squared << 
               ", " << rhs.count << "] ";
};

auto strings_to_string_views(std::vector<std::string>& input_strings) {
  auto all_valid =
    cudf::test::make_counting_transform_iterator(0, [](auto i) { return true; });
  std::vector<char> chars;
  std::vector<int32_t> offsets;
  std::tie(chars, offsets) = 
    cudf::test::detail::make_chars_and_offsets(
      input_strings.begin(), input_strings.end(), all_valid);
  thrust::device_vector<char> dev_chars(chars);
  char* c_start = thrust::raw_pointer_cast(dev_chars.data());

 // calculate the expected value by CPU. (but contains device pointers)
  std::vector<cudf::string_view> replaced_array(input_strings.size());
  std::transform(thrust::counting_iterator<size_t>(0),
                 thrust::counting_iterator<size_t>(replaced_array.size()),
                 replaced_array.begin(), [c_start, offsets](auto i) {
                   return cudf::string_view(c_start + offsets[i],
                                            offsets[i + 1] - offsets[i]);
                 });
  return std::make_tuple(std::move(dev_chars), replaced_array);
}

// ---------------------------------------------------------------------------

template <typename T>
struct IteratorTest : public cudf::test::BaseFixture
{
  // iterator test case which uses cub
  template <typename InputIterator, typename T_output>
  void iterator_test_cub(T_output expected, InputIterator d_in, int num_items)
  {
    T_output init{0};
    thrust::device_vector<T_output> dev_result(1, init);

    void* d_temp_storage = NULL;
    size_t temp_storage_bytes = 0;

    hipcub::DeviceReduce::Reduce(d_temp_storage, temp_storage_bytes, d_in,
                              dev_result.begin(), num_items,
                              thrust::minimum<T_output>{},
                              init);
    // Allocate temporary storage
    RMM_TRY(RMM_ALLOC(&d_temp_storage, temp_storage_bytes, 0));

    // Run reduction
    hipcub::DeviceReduce::Reduce(d_temp_storage, temp_storage_bytes, d_in,
                              dev_result.begin(), num_items,
                              thrust::minimum<T_output>{},
                              init);

    evaluate(expected, dev_result, "cub test");
  }

  // iterator test case which uses thrust
  template <typename InputIterator, typename T_output>
  void iterator_test_thrust(std::vector<T_output>& expected, InputIterator d_in, int num_items) 
  {
    InputIterator d_in_last = d_in + num_items;
    EXPECT_EQ(thrust::distance(d_in, d_in_last), num_items);
    thrust::device_vector<T_output> dev_expected(expected);

    // Can't use this because time_point make_pair bug in libcudacxx
    // bool result = thrust::equal(thrust::device, d_in, d_in_last, dev_expected.begin());
    bool result = thrust::transform_reduce(thrust::device,
        thrust::make_zip_iterator(thrust::make_tuple(d_in, dev_expected.begin())),
        thrust::make_zip_iterator(thrust::make_tuple(d_in_last, dev_expected.end())),
        [] __device__(auto it) { return (thrust::get<0>(it)) == T_output(thrust::get<1>(it)); },
        true,
        thrust::logical_and<bool>());
    #ifndef NDEBUG
    thrust::device_vector<bool> vec(expected.size(), false);
    thrust::transform(thrust::device,
        thrust::make_zip_iterator(thrust::make_tuple(d_in, dev_expected.begin())),
        thrust::make_zip_iterator(thrust::make_tuple(d_in_last, dev_expected.end())),
        vec.begin(),
        [] __device__(auto it) { return (thrust::get<0>(it)) == T_output(thrust::get<1>(it)); }
        );
    thrust::copy(vec.begin(), vec.end(), std::ostream_iterator<bool>(std::cout, " "));
    std::cout<<std::endl;
    #endif

    EXPECT_TRUE(result) << "thrust test";
  }

  template <typename T_output>
  void evaluate(T_output expected, thrust::device_vector<T_output>& dev_result, const char* msg = nullptr) 
  {
    thrust::host_vector<T_output> hos_result(dev_result);

    EXPECT_EQ(expected, hos_result[0]) << msg;
    std::cout << "Done: expected <" << msg << "> = "
      //<< hos_result[0] //TODO uncomment after time_point ostream operator<< 
      << std::endl;
  }

  template <typename T_output>
  void values_equal_test(std::vector<T_output>& expected, const cudf::column_device_view& col)
  {
    if (col.nullable()) {
      auto it_dev = cudf::experimental::detail::make_null_replacement_iterator(col, T_output{0});
      iterator_test_thrust(expected, it_dev, col.size());
    } else {
      auto it_dev = col.begin<T_output>();
      iterator_test_thrust(expected, it_dev, col.size());
    }
  }
};

using TestingTypes = cudf::test::AllTypes;

TYPED_TEST_CASE(IteratorTest, TestingTypes);

// tests for non-null iterator (pointer of device array)
TYPED_TEST(IteratorTest, non_null_iterator) {
  using T = TypeParam;
  std::vector<T> hos_array({0, 6, 0, -14, 13, 64, -13, -20, 45});
  thrust::device_vector<T> dev_array(hos_array);

  // calculate the expected value by CPU.
  std::vector<T> replaced_array(hos_array);

  // driven by iterator as a pointer of device array.
  // FIXME: compilation error for cudf::experimental::bool8
  // auto it_dev = dev_array.begin();
  // this->iterator_test_thrust(replaced_array, it_dev, dev_array.size());
  // this->iterator_test_cub(expected_value, it_dev, dev_array.size());

  // test column input
  cudf::test::fixed_width_column_wrapper<T> w_col(hos_array.begin(),
                                                  hos_array.end());
  this->values_equal_test(replaced_array, *cudf::column_device_view::create(w_col));
}

// Tests for null input iterator (column with null bitmap)
// Actually, we can use cub for reduction with nulls without creating custom
// kernel or multiple steps. We may accelarate the reduction for a column using
// cub
TYPED_TEST(IteratorTest, null_iterator) {
  using T = TypeParam;
  T init = T{0};
  // data and valid arrays
  std::vector<T> host_values({0, 6, 0, -14, 13, 64, -13, -20, 45});
  std::vector<bool> host_bools({1, 1, 0, 1, 1, 1, 0, 1, 1});

  // create a column with bool vector
  cudf::test::fixed_width_column_wrapper<T> w_col(host_values.begin(), host_values.end(),
                                                  host_bools.begin());
  auto d_col = cudf::column_device_view::create(w_col);

  // calculate the expected value by CPU.
  std::vector<T> replaced_array(host_values.size());
  std::transform(host_values.begin(), host_values.end(), host_bools.begin(),
                 replaced_array.begin(),
                 [&](T x, bool b) { return (b) ? x : init; });
  T expected_value =
      *std::min_element(replaced_array.begin(), replaced_array.end());
  // TODO uncomment after time_point ostream operator<<
  // std::cout << "expected <null_iterator> = " << expected_value << std::endl;

  // GPU test
  auto it_dev = cudf::experimental::detail::make_null_replacement_iterator(*d_col, T{0});
  this->iterator_test_cub(expected_value, it_dev, d_col->size());
  this->values_equal_test(replaced_array, *d_col);
}

// Tests up cast reduction with null iterator.
// The up cast iterator will be created by transform_iterator and
// cudf::experimental::detail::make_null_replacement_iterator(col, T{0})
TYPED_TEST(IteratorTest, null_iterator_upcast) {
  const int column_size{1000};
  using T = int8_t;
  using T_upcast = int64_t;
  T init{0};

  // data and valid arrays
  std::vector<T> host_values(column_size);
  std::generate(host_values.begin(), host_values.end(),
                []() { return static_cast<T>(random_int<T>(-128, 127)); });
  std::vector<bool> host_bools(column_size);
  std::generate(host_bools.begin(), host_bools.end(),
                []() { return static_cast<bool>(random_bool()); });

  cudf::test::fixed_width_column_wrapper<T> w_col(host_values.begin(), host_values.end(),
                                                  host_bools.begin());
  auto d_col = cudf::column_device_view::create(w_col);

  // calculate the expected value by CPU.
  std::vector<T> replaced_array(d_col->size());
  std::transform(host_values.begin(), host_values.end(), host_bools.begin(),
                 replaced_array.begin(),
                 [&](T x, bool b) { return (b) ? x : init; });
  T_upcast expected_value =
      *std::min_element(replaced_array.begin(), replaced_array.end());
  // std::cout << "expected <null_iterator> = " << expected_value << std::endl;

  // GPU test
  auto it_dev = cudf::experimental::detail::make_null_replacement_iterator(*d_col, T{0});
  auto it_dev_upcast =
      thrust::make_transform_iterator(it_dev, thrust::identity<T_upcast>());
  this->iterator_test_thrust(replaced_array, it_dev_upcast, d_col->size());
  this->iterator_test_cub(expected_value, it_dev, d_col->size());
}

// Tests for square input iterator using helper strcut
// `cudf::transformer_squared<T, T_upcast>` The up cast iterator will be created
// by make_transform_iterator(
//        cudf::experimental::detail::make_null_replacement_iterator(col, T{0}), 
//        cudf::detail::transformer_squared<T_upcast>)
TYPED_TEST(IteratorTest, null_iterator_square) {
  const int column_size{1000};
  using T = int8_t;
  using T_upcast = int64_t;
  T init{0};
  cudf::transformer_squared<T_upcast> transformer{};

  // data and valid arrays
  std::vector<T> host_values(column_size);
  std::generate(host_values.begin(), host_values.end(),
                []() { return static_cast<T>(random_int(-128, 128)); });
  std::vector<bool> host_bools(column_size);
  std::generate(host_bools.begin(), host_bools.end(),
                []() { return static_cast<bool>(random_bool()); });

  cudf::test::fixed_width_column_wrapper<T> w_col(host_values.begin(), host_values.end(),
                                                  host_bools.begin());
  auto d_col = cudf::column_device_view::create(w_col);

  // calculate the expected value by CPU.
  std::vector<T_upcast> replaced_array(d_col->size());
  std::transform(host_values.begin(), host_values.end(), host_bools.begin(),
                 replaced_array.begin(),
                 [&](T x, bool b) { return (b) ? x * x : init; });
  T_upcast expected_value =
      *std::min_element(replaced_array.begin(), replaced_array.end());
  // std::cout << "expected <null_iterator> = " << expected_value << std::endl;

  // GPU test
  auto it_dev = cudf::experimental::detail::make_null_replacement_iterator(*d_col, T{0});
  auto it_dev_upcast = thrust::make_transform_iterator(it_dev, thrust::identity<T_upcast>());
  auto it_dev_squared = thrust::make_transform_iterator(it_dev_upcast, transformer);
  this->iterator_test_thrust(replaced_array, it_dev_squared, d_col->size());
  this->iterator_test_cub(expected_value, it_dev_squared, d_col->size());
}

TYPED_TEST(IteratorTest, large_size_reduction) {
  using T = TypeParam;

  const int column_size{1000000};
  const T init{0};

  // data and valid arrays
  std::vector<T> host_values(column_size);
  std::generate(host_values.begin(), host_values.end(),
                []() { return static_cast<T>(random_int(-128, 128)); });
  std::vector<bool> host_bools(column_size);
  std::generate(host_bools.begin(), host_bools.end(),
                []() { return static_cast<bool>(random_bool()); });

  cudf::test::fixed_width_column_wrapper<TypeParam> w_col(
      host_values.begin(), host_values.end(), host_bools.begin());
  auto d_col = cudf::column_device_view::create(w_col);

  // calculate by cudf::reduce
  std::vector<T> replaced_array(d_col->size());
  std::transform(host_values.begin(), host_values.end(), host_bools.begin(),
                 replaced_array.begin(),
                 [&](T x, bool b) { return (b) ? x : init; });
  T expected_value = *std::min_element(replaced_array.begin(), replaced_array.end());
  // std::cout << "expected <null_iterator> = " << expected_value << std::endl;

  // GPU test
  auto it_dev = cudf::experimental::detail::make_null_replacement_iterator(*d_col, init);
  this->iterator_test_thrust(replaced_array, it_dev, d_col->size());
  this->iterator_test_cub(expected_value, it_dev, d_col->size());
}

// Transformers and Operators for pair_iterator test
template<typename ElementType>
struct transformer_pair_meanvar
{
    using ResultType = thrust::pair<cudf::meanvar<ElementType>, bool>;

    CUDA_HOST_DEVICE_CALLABLE
    ResultType operator()(thrust::pair<ElementType, bool> const& pair)
    {
        ElementType v = pair.first;
        return {{v, static_cast<ElementType>(v*v), (pair.second)? 1 : 0 }, pair.second};
    };
};

struct sum_if_not_null {
  template <typename T>
  CUDA_HOST_DEVICE_CALLABLE thrust::pair<T, bool> operator()( const thrust::pair<T, bool>& lhs, const thrust::pair<T, bool>& rhs) {
    if (lhs.second & rhs.second)
      return {lhs.first+rhs.first, true};
    else if (lhs.second)
      return {lhs};
    else 
      return {rhs};
  }
};

template <typename T>
struct PairIteratorTest : public cudf::test::BaseFixture {};
TYPED_TEST_CASE(PairIteratorTest, cudf::test::NumericTypes);
// TODO: enable this test also at __CUDACC_DEBUG__
// This test causes fatal compilation error only at device debug mode.
// Workaround: exclude this test only at device debug mode.
#if !defined(__CUDACC_DEBUG__)
// This test computes `count`, `sum`, `sum_of_squares` at a single reduction call.
// It would be useful for `var`, `std` operation
TYPED_TEST(PairIteratorTest, mean_var_output) {
  using T = TypeParam;
  using T_output = cudf::meanvar<T>;
  transformer_pair_meanvar<T> transformer{};

  const int column_size{5000};
  const T init{0};

  // data and valid arrays
  std::vector<T> host_values(column_size);
  std::generate(host_values.begin(), host_values.end(),
                []() { return static_cast<T>(random_int(-128, 128)); });

  std::vector<bool> host_bools(column_size);
  std::generate(host_bools.begin(), host_bools.end(),
                []() { return static_cast<bool>(random_bool()); });

  cudf::test::fixed_width_column_wrapper<TypeParam> w_col(
      host_values.begin(), host_values.end(), host_bools.begin());
  auto d_col = cudf::column_device_view::create(w_col);

  // calculate expected values by CPU
  T_output expected_value;

  expected_value.count = d_col->size() - d_col->null_count();

  std::vector<T> replaced_array(d_col->size());
  std::transform(host_values.begin(), host_values.end(), host_bools.begin(),
                 replaced_array.begin(),
                 [&](T x, bool b) { return (b) ? static_cast<T>(x) : init; });

  expected_value.count = d_col->size() - d_col->null_count();
  expected_value.value = std::accumulate(replaced_array.begin(),
                                         replaced_array.end(), T{0});
  expected_value.value_squared =
      std::accumulate(replaced_array.begin(), replaced_array.end(), T{0},
                      [](T acc, T i) { return acc + i * i; });

  std::cout << "expected <mixed_output> = " << expected_value << std::endl;

  // GPU test
  auto it_dev = d_col->pair_begin<T, true>();
  auto it_dev_squared = thrust::make_transform_iterator(it_dev, transformer);
  auto result = thrust::reduce( it_dev_squared, it_dev_squared+ d_col->size(), thrust::make_pair(T_output{}, true), sum_if_not_null{} );
  EXPECT_EQ(expected_value, result.first) << "pair iterator reduction sum";
}
#endif

TYPED_TEST(IteratorTest, error_handling) {
  using T = TypeParam;
  std::vector<T> hos_array({0, 6, 0, -14, 13, 64, -13, -20, 45});
  std::vector<bool> host_bools({1, 1, 0, 1, 1, 1, 0, 1, 1});

  cudf::test::fixed_width_column_wrapper<T> w_col_no_null(hos_array.begin(),
                                                          hos_array.end());
  cudf::test::fixed_width_column_wrapper<T> w_col_null(hos_array.begin(),
                                                       hos_array.end(),
                                                       host_bools.begin());
  auto d_col_no_null = cudf::column_device_view::create(w_col_no_null);
  auto d_col_null = cudf::column_device_view::create(w_col_null);

  // expects error: data type mismatch
  if (!(std::is_same<T, double>::value)) {
    CUDF_EXPECT_THROW_MESSAGE((d_col_null->begin<double>()),
                              "the data type mismatch");
  }
  // expects error: data type mismatch
  if (!(std::is_same<T, float>::value)) {
    CUDF_EXPECT_THROW_MESSAGE((cudf::experimental::detail::make_null_replacement_iterator(*d_col_null, float{0})),
                              "the data type mismatch");
  }

  CUDF_EXPECT_THROW_MESSAGE((cudf::experimental::detail::make_null_replacement_iterator(*d_col_no_null, T{0})),
                            "Unexpected non-nullable column.");

  CUDF_EXPECT_THROW_MESSAGE((d_col_null->begin<T>()),
                            "Unexpected column with nulls.");

  CUDF_EXPECT_THROW_MESSAGE((d_col_no_null->pair_begin<T, true>()),
                            "Unexpected non-nullable column.");
  CUDF_EXPECT_NO_THROW((d_col_null->pair_begin<T, false>()));
  CUDF_EXPECT_NO_THROW((d_col_null->pair_begin<T, true>()));

  //scalar iterator
  using ScalarType = cudf::experimental::scalar_type_t<T>;
  std::unique_ptr<cudf::scalar> s(new ScalarType{T{1}, false});
  CUDF_EXPECT_THROW_MESSAGE((cudf::experimental::detail::make_scalar_iterator<T>(*s)),
                            "the scalar value must be valid");
  CUDF_EXPECT_NO_THROW((cudf::experimental::detail::make_pair_iterator<T>(*s)));
  // expects error: data type mismatch
  if (!(std::is_same<T, double>::value)) {
    CUDF_EXPECT_THROW_MESSAGE((cudf::experimental::detail::make_scalar_iterator<double>(*s)),
                              "the data type mismatch");
    CUDF_EXPECT_THROW_MESSAGE((cudf::experimental::detail::make_pair_iterator<double>(*s)),
                              "the data type mismatch");
  }
}

struct StringIteratorTest :  public IteratorTest<cudf::string_view> { 
};

TEST_F(StringIteratorTest, string_view_null_iterator ) {
  using T = cudf::string_view;
  // T init = T{"", 0};
  std::string zero("zero");
  // the char data has to be in GPU
  thrust::device_vector<char> initmsg(zero.begin(), zero.end());
  T init = T{initmsg.data().get(), int(initmsg.size())};

  // data and valid arrays
  std::vector<std::string> host_values({"one", "two", "three", "four", "five", "six", "eight", "nine"});
  std::vector<bool> host_bools({1, 1, 0, 1, 1, 1, 0, 1, 1});

  // replace nulls in CPU
  std::vector<std::string> replaced_strings(host_values.size());
  std::transform(host_values.begin(), host_values.end(), host_bools.begin(),
                 replaced_strings.begin(),
                 [zero](auto s, auto b) { return b ? s : zero; });

  thrust::device_vector<char> dev_chars;
  std::vector<T> replaced_array(host_values.size());
  std::tie(dev_chars, replaced_array) = strings_to_string_views(replaced_strings);

  // create a column with bool vector
  cudf::test::strings_column_wrapper w_col(host_values.begin(), host_values.end(),
                                           host_bools.begin());
  auto d_col = cudf::column_device_view::create(w_col);
 
  // GPU test
  auto it_dev = cudf::experimental::detail::make_null_replacement_iterator(*d_col, init);
  this->iterator_test_thrust(replaced_array, it_dev, host_values.size());
  // this->values_equal_test(replaced_array, *d_col); //string_view{0} is invalid
}

TEST_F(StringIteratorTest, string_view_no_null_iterator ) {
  using T = cudf::string_view;
  // T init = T{"", 0};
  std::string zero("zero");
  // the char data has to be in GPU
  thrust::device_vector<char> initmsg(zero.begin(), zero.end());
  T init = T{initmsg.data().get(), int(initmsg.size())};

  // data array
  std::vector<std::string> host_values({"one", "two", "three", "four", "five", "six", "eight", "nine"});

  thrust::device_vector<char> dev_chars;
  std::vector<T> all_array(host_values.size());
  std::tie(dev_chars, all_array) = strings_to_string_views(host_values);

  // create a column with bool vector
  cudf::test::strings_column_wrapper w_col(host_values.begin(), host_values.end());
  auto d_col = cudf::column_device_view::create(w_col);
 
  // GPU test
  auto it_dev = d_col->begin<T>();
  this->iterator_test_thrust(all_array, it_dev, host_values.size());
}

TYPED_TEST(IteratorTest, nonull_pair_iterator) {
  using T = TypeParam;
  // data and valid arrays
  std::vector<T> host_values({0, 6, 0, -14, 13, 64, -13, -20, 45});

  // create a column
  cudf::test::fixed_width_column_wrapper<T> w_col(host_values.begin(), host_values.end());
  auto d_col = cudf::column_device_view::create(w_col);
 
  // calculate the expected value by CPU.
  std::vector<thrust::pair<T,bool> > replaced_array(host_values.size());
  std::transform(host_values.begin(), host_values.end(), 
                 replaced_array.begin(),
                 [](auto s) { return thrust::make_pair(s, true); });

  // GPU test
  auto it_dev = d_col->pair_begin<T, false>();
  this->iterator_test_thrust(replaced_array, it_dev, host_values.size());
}

TYPED_TEST(IteratorTest, null_pair_iterator) {
  using T = TypeParam;
  // data and valid arrays
  std::vector<T> host_values({0, 6, 0, -14, 13, 64, -13, -20, 45});
  std::vector<bool> host_bools({1, 1, 0, 1, 1, 1, 0, 1, 1});

  // create a column with bool vector
  cudf::test::fixed_width_column_wrapper<T> w_col(host_values.begin(), host_values.end(),
                                           host_bools.begin());
  auto d_col = cudf::column_device_view::create(w_col);
 
  // calculate the expected value by CPU.
  std::vector<thrust::pair<T,bool> > value_and_validity(host_values.size());
  std::transform(host_values.begin(), host_values.end(), host_bools.begin(),
                 value_and_validity.begin(),
                 [](auto s, auto b) { return thrust::pair<T, bool>{s, b}; });
  std::vector<thrust::pair<T,bool> > value_all_valid(host_values.size());
  std::transform(host_values.begin(), host_values.end(), host_bools.begin(),
                 value_all_valid.begin(),
                 [](auto s, auto b) { return thrust::pair<T, bool>{s, true}; });

  // GPU test
  auto it_dev = d_col->pair_begin<T, true>();
  this->iterator_test_thrust(value_and_validity, it_dev, host_values.size());

  auto it_hasnonull_dev = d_col->pair_begin<T, false>();
  this->iterator_test_thrust(value_all_valid, it_hasnonull_dev, host_values.size());
  
  auto itb_dev = cudf::experimental::detail::make_validity_iterator(*d_col);
  this->iterator_test_thrust(host_bools, itb_dev, host_values.size());
}

TYPED_TEST(IteratorTest, scalar_iterator) {
  using T = TypeParam;
  T init = static_cast<T>(random_int(-128, 128));
  // data and valid arrays
  std::vector<T> host_values(100, init);
  std::vector<bool> host_bools(100, true);

  // create a scalar
  using ScalarType = cudf::experimental::scalar_type_t<T>;
  std::unique_ptr<cudf::scalar> s(new ScalarType{init, true});
 
  // calculate the expected value by CPU.
  std::vector<thrust::pair<T,bool> > value_and_validity(host_values.size());
  std::transform(host_values.begin(), host_values.end(), host_bools.begin(),
                 value_and_validity.begin(),
                 [](auto v, auto b) { return thrust::pair<T, bool>{v, b}; });

  // GPU test
  auto it_dev = cudf::experimental::detail::make_scalar_iterator<T>(*s);
  this->iterator_test_thrust(host_values, it_dev, host_values.size());

  auto it_pair_dev = cudf::experimental::detail::make_pair_iterator<T>(*s);
  this->iterator_test_thrust(value_and_validity, it_pair_dev, host_values.size());
}

TYPED_TEST(IteratorTest, null_scalar_iterator) {
  using T = TypeParam;
  T init = static_cast<T>(random_int(-128, 128));
  // data and valid arrays
  std::vector<T> host_values(100, init);
  std::vector<bool> host_bools(100, true);

  // create a scalar
  using ScalarType = cudf::experimental::scalar_type_t<T>;
  std::unique_ptr<cudf::scalar> s(new ScalarType{init, true});
 
  // calculate the expected value by CPU.
  std::vector<thrust::pair<T,bool> > value_and_validity(host_values.size());
  std::transform(host_values.begin(), host_values.end(), host_bools.begin(),
                 value_and_validity.begin(),
                 [](auto v, auto b) { return thrust::pair<T, bool>{v, b}; });

  // GPU test
  auto it_pair_dev = cudf::experimental::detail::make_pair_iterator<T>(*s);
  this->iterator_test_thrust(value_and_validity, it_pair_dev, host_values.size());
}