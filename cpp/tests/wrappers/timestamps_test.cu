/*
 * Copyright (c) 2019, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <cudf/column/column_device_view.cuh>
#include <cudf/column/column_factories.hpp>
#include <cudf/legacy/binaryop.hpp>
#include <cudf/wrappers/timestamps.hpp>

#include <tests/utilities/base_fixture.hpp>
#include <tests/utilities/timestamp_utilities.cuh>
#include <tests/utilities/type_lists.hpp>
#include "cudf/column/column_view.hpp"
#include "cudf/types.hpp"
#include "tests/utilities/column_utilities.hpp"
#include "tests/utilities/column_wrapper.hpp"

#include <gmock/gmock.h>

template <typename T>
struct TimestampColumnTest : public cudf::test::BaseFixture {
  hipStream_t stream() { return hipStream_t(0); }
  cudf::size_type size() { return cudf::size_type(100); }
  cudf::data_type type() { return cudf::data_type{cudf::experimental::type_to_id<T>()}; }
};

template <typename Timestamp>
struct compare_timestamp_elements_to_primitive_representation {
  cudf::column_device_view primitives;
  cudf::column_device_view timestamps;

  compare_timestamp_elements_to_primitive_representation(cudf::column_device_view& _primitives,
                                                         cudf::column_device_view& _timestamps)
    : primitives(_primitives), timestamps(_timestamps)
  {
  }

  __host__ __device__ bool operator()(const int32_t element_index)
  {
    using Primitive = typename Timestamp::rep;
    auto primitive  = primitives.element<Primitive>(element_index);
    auto timestamp  = timestamps.element<Timestamp>(element_index);
    return primitive == timestamp.time_since_epoch().count();
  }
};

TYPED_TEST_CASE(TimestampColumnTest, cudf::test::TimestampTypes);

TYPED_TEST(TimestampColumnTest, TimestampDurationsMatchPrimitiveRepresentation)
{
  using T   = TypeParam;
  using Rep = typename T::rep;
  using namespace cudf::test;
  using namespace simt::std::chrono;

  auto start = milliseconds(-2500000000000);  // Sat, 11 Oct 1890 19:33:20 GMT
  auto stop_ = milliseconds(2500000000000);   // Mon, 22 Mar 2049 04:26:40 GMT
  auto timestamp_col =
    generate_timestamps<T>(this->size(), time_point_ms(start), time_point_ms(stop_));

  // rount-trip through the host to copy `timestamp_col` values
  // to a new fixed_width_column_wrapper `primitive_col`
  // When C++17, use structured bindings
  thrust::host_vector<Rep> timestamp_col_data;
  std::vector<cudf::bitmask_type> timestamp_col_mask;
  std::tie(timestamp_col_data, timestamp_col_mask) = to_host<Rep>(timestamp_col);

  auto primitive_col =
    fixed_width_column_wrapper<Rep>(timestamp_col_data.begin(), timestamp_col_data.end());

  thrust::device_vector<int32_t> indices(this->size());
  thrust::sequence(indices.begin(), indices.end());
  EXPECT_TRUE(thrust::all_of(indices.begin(),
                             indices.end(),
                             compare_timestamp_elements_to_primitive_representation<T>{
                               *cudf::column_device_view::create(primitive_col),
                               *cudf::column_device_view::create(timestamp_col)}));
}

template <typename Timestamp>
struct compare_timestamp_elements {
  gdf_binary_operator comp;
  cudf::column_device_view lhs;
  cudf::column_device_view rhs;

  compare_timestamp_elements(gdf_binary_operator _comp,
                             cudf::column_device_view& _lhs,
                             cudf::column_device_view& _rhs)
    : comp(_comp), lhs(_lhs), rhs(_rhs)
  {
  }

  __host__ __device__ bool operator()(const int32_t element_index)
  {
    auto lhs_elt = lhs.element<Timestamp>(element_index);
    auto rhs_elt = rhs.element<Timestamp>(element_index);
    switch (comp) {
      case GDF_LESS: return lhs_elt < rhs_elt;
      case GDF_GREATER: return lhs_elt > rhs_elt;
      case GDF_LESS_EQUAL: return lhs_elt <= rhs_elt;
      case GDF_GREATER_EQUAL: return lhs_elt >= rhs_elt;
      default: return false;
    }
  }
};

TYPED_TEST(TimestampColumnTest, TimestampsCanBeComparedInDeviceCode)
{
  using T = TypeParam;
  using namespace cudf::test;
  using namespace simt::std::chrono;

  auto start_lhs = milliseconds(-2500000000000);  // Sat, 11 Oct 1890 19:33:20 GMT
  auto start_rhs = milliseconds(-2400000000000);  // Tue, 12 Dec 1893 05:20:00 GMT
  auto stop_lhs_ = milliseconds(2500000000000);   // Mon, 22 Mar 2049 04:26:40 GMT
  auto stop_rhs_ = milliseconds(2600000000000);   // Wed, 22 May 2052 14:13:20 GMT

  auto timestamp_lhs_col =
    generate_timestamps<T>(this->size(), time_point_ms(start_lhs), time_point_ms(stop_lhs_));

  auto timestamp_rhs_col =
    generate_timestamps<T>(this->size(), time_point_ms(start_rhs), time_point_ms(stop_rhs_));

  thrust::device_vector<int32_t> indices(this->size());
  thrust::sequence(indices.begin(), indices.end());

  EXPECT_TRUE(thrust::all_of(
    indices.begin(),
    indices.end(),
    compare_timestamp_elements<TypeParam>{GDF_LESS,
                                          *cudf::column_device_view::create(timestamp_lhs_col),
                                          *cudf::column_device_view::create(timestamp_rhs_col)}));

  EXPECT_TRUE(thrust::all_of(
    indices.begin(),
    indices.end(),
    compare_timestamp_elements<TypeParam>{GDF_GREATER,
                                          *cudf::column_device_view::create(timestamp_rhs_col),
                                          *cudf::column_device_view::create(timestamp_lhs_col)}));

  EXPECT_TRUE(thrust::all_of(
    indices.begin(),
    indices.end(),
    compare_timestamp_elements<TypeParam>{GDF_LESS_EQUAL,
                                          *cudf::column_device_view::create(timestamp_lhs_col),
                                          *cudf::column_device_view::create(timestamp_lhs_col)}));

  EXPECT_TRUE(thrust::all_of(
    indices.begin(),
    indices.end(),
    compare_timestamp_elements<TypeParam>{GDF_GREATER_EQUAL,
                                          *cudf::column_device_view::create(timestamp_rhs_col),
                                          *cudf::column_device_view::create(timestamp_rhs_col)}));
}

TYPED_TEST(TimestampColumnTest, TimestampFactoryNullMaskAsParm)
{
  rmm::device_buffer null_mask{create_null_mask(this->size(), cudf::mask_state::ALL_NULL)};
  auto column =
    cudf::make_timestamp_column(cudf::data_type{cudf::experimental::type_to_id<TypeParam>()},
                                this->size(),
                                null_mask,
                                this->size(),
                                this->stream(),
                                this->mr());
  EXPECT_EQ(column->type(), cudf::data_type{cudf::experimental::type_to_id<TypeParam>()});
  EXPECT_EQ(column->size(), this->size());
  EXPECT_EQ(this->size(), column->null_count());
  EXPECT_TRUE(column->nullable());
  EXPECT_TRUE(column->has_nulls());
  EXPECT_EQ(0, column->num_children());
}

TYPED_TEST(TimestampColumnTest, TimestampFactoryNullMaskAsEmptyParm)
{
  rmm::device_buffer null_mask{};
  auto column =
    cudf::make_timestamp_column(cudf::data_type{cudf::experimental::type_to_id<TypeParam>()},
                                this->size(),
                                null_mask,
                                0,
                                this->stream(),
                                this->mr());
  EXPECT_EQ(column->type(), cudf::data_type{cudf::experimental::type_to_id<TypeParam>()});
  EXPECT_EQ(column->size(), this->size());
  EXPECT_EQ(0, column->null_count());
  EXPECT_FALSE(column->nullable());
  EXPECT_FALSE(column->has_nulls());
  EXPECT_EQ(0, column->num_children());
}

CUDF_TEST_PROGRAM_MAIN()
