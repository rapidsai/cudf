/*
 * Copyright (c) 2021-2022, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <cudf/column/column.hpp>
#include <cudf/column/column_factories.hpp>
#include <cudf/column/column_view.hpp>
#include <cudf/io/parquet.hpp>
#include <cudf/transform.hpp>
#include <cudf/types.hpp>
#include <cudf_test/base_fixture.hpp>
#include <cudf_test/column_utilities.hpp>
#include <cudf_test/column_wrapper.hpp>
#include <cudf_test/type_lists.hpp>

#include <rmm/exec_policy.hpp>

#include <thrust/fill.h>
#include <thrust/functional.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/tabulate.h>
#include <thrust/transform.h>

using namespace cudf;

template <typename T>
struct RowBitCountTyped : public cudf::test::BaseFixture {
};

TYPED_TEST_SUITE(RowBitCountTyped, cudf::test::FixedWidthTypes);

TYPED_TEST(RowBitCountTyped, SimpleTypes)
{
  using T = TypeParam;

  auto col = cudf::make_fixed_width_column(data_type{type_to_id<T>()}, 16);

  table_view t({*col});
  auto result = cudf::row_bit_count(t);

  // expect size of the type per row
  auto expected = make_fixed_width_column(data_type{type_id::INT32}, 16);
  cudf::mutable_column_view mcv(*expected);
  thrust::fill(rmm::exec_policy(cudf::default_stream_value),
               mcv.begin<size_type>(),
               mcv.end<size_type>(),
               sizeof(device_storage_type_t<T>) * CHAR_BIT);

  CUDF_TEST_EXPECT_COLUMNS_EQUAL(*expected, *result);
}

TYPED_TEST(RowBitCountTyped, SimpleTypesWithNulls)
{
  using T = TypeParam;

  auto iter   = thrust::make_counting_iterator(0);
  auto valids = cudf::detail::make_counting_transform_iterator(
    0, [](int i) { return i % 2 == 0 ? true : false; });
  cudf::test::fixed_width_column_wrapper<T> col(iter, iter + 16, valids);

  table_view t({col});
  auto result = cudf::row_bit_count(t);

  // expect size of the type + 1 bit per row
  auto expected = make_fixed_width_column(data_type{type_id::INT32}, 16);
  cudf::mutable_column_view mcv(*expected);
  thrust::fill(rmm::exec_policy(cudf::default_stream_value),
               mcv.begin<size_type>(),
               mcv.end<size_type>(),
               (sizeof(device_storage_type_t<T>) * CHAR_BIT) + 1);

  CUDF_TEST_EXPECT_COLUMNS_EQUAL(*expected, *result);
}

template <typename T>
std::pair<std::unique_ptr<column>, std::unique_ptr<column>> build_list_column()
{
  using LCW                     = cudf::test::lists_column_wrapper<T, int>;
  constexpr size_type type_size = sizeof(device_storage_type_t<T>) * CHAR_BIT;

  // {
  //  {{1, 2}, {3, 4, 5}},
  //  {{}},
  //  {LCW{10}},
  //  {{6, 7, 8}, {9}},
  //  {{-1, -2}, {-3, -4}},
  //  {{-5, -6, -7}, {-8, -9}}
  // }
  cudf::test::fixed_width_column_wrapper<T> values{
    1, 2, 3, 4, 5, 10, 6, 7, 8, 9, -1, -2, -3, -4, -5, -6, -7, -8, -9};
  cudf::test::fixed_width_column_wrapper<offset_type> inner_offsets{
    0, 2, 5, 6, 9, 10, 12, 14, 17, 19};
  auto inner_list = cudf::make_lists_column(9, inner_offsets.release(), values.release(), 0, {});
  cudf::test::fixed_width_column_wrapper<offset_type> outer_offsets{0, 2, 2, 3, 5, 7, 9};
  auto list = cudf::make_lists_column(6, outer_offsets.release(), std::move(inner_list), 0, {});

  // expected size = (num rows at level 1 + num_rows at level 2) + # values in the leaf
  cudf::test::fixed_width_column_wrapper<size_type> expected{
    ((4 + 8) * CHAR_BIT) + (type_size * 5),
    ((4 + 0) * CHAR_BIT) + (type_size * 0),
    ((4 + 4) * CHAR_BIT) + (type_size * 1),
    ((4 + 8) * CHAR_BIT) + (type_size * 4),
    ((4 + 8) * CHAR_BIT) + (type_size * 4),
    ((4 + 8) * CHAR_BIT) + (type_size * 5)};

  return {std::move(list), expected.release()};
}

TYPED_TEST(RowBitCountTyped, Lists)
{
  using T = TypeParam;

  auto [col, expected_sizes] = build_list_column<T>();

  table_view t({*col});
  auto result = cudf::row_bit_count(t);

  CUDF_TEST_EXPECT_COLUMNS_EQUAL(*expected_sizes, *result);
}

TYPED_TEST(RowBitCountTyped, ListsWithNulls)
{
  using T                       = TypeParam;
  using LCW                     = cudf::test::lists_column_wrapper<T, int>;
  constexpr size_type type_size = sizeof(device_storage_type_t<T>) * CHAR_BIT;

  // {
  //  {{1, 2}, {3, null, 5}},
  //  {{}},
  //  {LCW{10}},
  //  {{null, 7, null}, null},
  // }
  cudf::test::fixed_width_column_wrapper<T> values{{1, 2, 3, 4, 5, 10, 6, 7, 8},
                                                   {1, 1, 1, 0, 1, 1, 0, 1, 0}};
  cudf::test::fixed_width_column_wrapper<offset_type> inner_offsets{0, 2, 5, 6, 9, 9};
  std::vector<bool> inner_list_validity{1, 1, 1, 1, 0};
  auto inner_list = cudf::make_lists_column(
    5,
    inner_offsets.release(),
    values.release(),
    1,
    cudf::test::detail::make_null_mask(inner_list_validity.begin(), inner_list_validity.end()));
  cudf::test::fixed_width_column_wrapper<offset_type> outer_offsets{0, 2, 2, 3, 5};
  auto list = cudf::make_lists_column(4, outer_offsets.release(), std::move(inner_list), 0, {});

  table_view t({*list});
  auto result = cudf::row_bit_count(t);

  // expected size = (num rows at level 1 + num_rows at level 2) + # values in the leaf + validity
  // where applicable
  cudf::test::fixed_width_column_wrapper<size_type> expected{
    ((4 + 8) * CHAR_BIT) + (type_size * 5) + 7,
    ((4 + 0) * CHAR_BIT) + (type_size * 0),
    ((4 + 4) * CHAR_BIT) + (type_size * 1) + 2,
    ((4 + 8) * CHAR_BIT) + (type_size * 3) + 5};
  CUDF_TEST_EXPECT_COLUMNS_EQUAL(expected, *result);
}

struct RowBitCount : public cudf::test::BaseFixture {
};

TEST_F(RowBitCount, Strings)
{
  std::vector<std::string> strings{"abc", "ï", "", "z", "bananas", "warp", "", "zing"};

  cudf::test::strings_column_wrapper col(strings.begin(), strings.end());

  table_view t({col});
  auto result = cudf::row_bit_count(t);

  // expect 1 offset (4 bytes) + length of string per row
  auto size_iter = cudf::detail::make_counting_transform_iterator(0, [&strings](int i) {
    return (static_cast<size_type>(strings[i].size()) + sizeof(offset_type)) * CHAR_BIT;
  });
  cudf::test::fixed_width_column_wrapper<size_type> expected(size_iter, size_iter + strings.size());

  CUDF_TEST_EXPECT_COLUMNS_EQUAL(expected, *result);
}

TEST_F(RowBitCount, StringsWithNulls)
{
  // clang-format off
  std::vector<std::string> strings { "daïs", "def", "", "z", "bananas", "warp", "", "zing" };
  std::vector<bool>        valids  {  1,      0,    0,  1,   0,          1,      1,  1 };
  // clang-format on

  cudf::test::strings_column_wrapper col(strings.begin(), strings.end(), valids.begin());

  table_view t({col});
  auto result = cudf::row_bit_count(t);

  // expect 1 offset (4 bytes) + (length of string, or 0 if null) + 1 validity bit per row
  auto size_iter = cudf::detail::make_counting_transform_iterator(0, [&strings, &valids](int i) {
    return ((static_cast<size_type>(valids[i] ? strings[i].size() : 0) + sizeof(offset_type)) *
            CHAR_BIT) +
           1;
  });
  cudf::test::fixed_width_column_wrapper<size_type> expected(size_iter, size_iter + strings.size());

  CUDF_TEST_EXPECT_COLUMNS_EQUAL(expected, *result);
}

namespace {

/**
 * @brief __device__ functor to multiply input by 2, defined out of line because __device__ lambdas
 * cannot be defined in a TEST_F().
 */
struct times_2 {
  int32_t __device__ operator()(int32_t i) const { return i * 2; }
};

}  // namespace

TEST_F(RowBitCount, StructsWithLists_RowsExceedingASingleBlock)
{
  // Tests that `row_bit_count()` can handle struct<list<int32_t>> with more
  // than max_block_size (256) rows.
  // With a large number of rows, computation spills to multiple thread-blocks,
  // thus exercising the branch-stack computation.
  // The contents of the input column aren't as pertinent to this test as the
  // column size. For what it's worth, it looks as follows:
  //   [ struct({0,1}), struct({2,3}), struct({4,5}), ... ]

  using namespace cudf;
  auto constexpr num_rows = 1024 * 2;  // Exceeding a block size.

  // List child column = {0, 1, 2, 3, 4, ..., 2*num_rows};
  auto ints      = make_numeric_column(data_type{type_id::INT32}, num_rows * 2);
  auto ints_view = ints->mutable_view();
  thrust::tabulate(rmm::exec_policy(cudf::default_stream_value),
                   ints_view.begin<int32_t>(),
                   ints_view.end<int32_t>(),
                   thrust::identity{});

  // List offsets = {0, 2, 4, 6, 8, ..., num_rows*2};
  auto list_offsets      = make_numeric_column(data_type{type_id::INT32}, num_rows + 1);
  auto list_offsets_view = list_offsets->mutable_view();
  thrust::tabulate(rmm::exec_policy(cudf::default_stream_value),
                   list_offsets_view.begin<offset_type>(),
                   list_offsets_view.end<offset_type>(),
                   times_2{});

  // List<int32_t> = {{0,1}, {2,3}, {4,5}, ..., {2*(num_rows-1), 2*num_rows-1}};
  auto lists_column = make_lists_column(num_rows, std::move(list_offsets), std::move(ints), 0, {});

  // Struct<List<int32_t>.
  auto struct_members = std::vector<std::unique_ptr<column>>{};
  struct_members.emplace_back(std::move(lists_column));
  auto structs_column = make_structs_column(num_rows, std::move(struct_members), 0, {});

  // Compute row_bit_count, and compare.
  auto row_bit_counts          = row_bit_count(table_view{{structs_column->view()}});
  auto expected_row_bit_counts = make_numeric_column(data_type{type_id::INT32}, num_rows);
  thrust::fill_n(rmm::exec_policy(cudf::default_stream_value),
                 expected_row_bit_counts->mutable_view().begin<int32_t>(),
                 num_rows,
                 CHAR_BIT * (2 * sizeof(int32_t) + sizeof(offset_type)));

  CUDF_TEST_EXPECT_COLUMNS_EQUIVALENT(row_bit_counts->view(), expected_row_bit_counts->view());
}

std::pair<std::unique_ptr<column>, std::unique_ptr<column>> build_struct_column()
{
  std::vector<bool> struct_validity{0, 1, 1, 1, 1, 0};
  std::vector<std::string> strings{"abc", "def", "", "z", "bananas", "daïs"};

  cudf::test::fixed_width_column_wrapper<float> col0{0, 1, 2, 3, 4, 5};
  cudf::test::fixed_width_column_wrapper<int16_t> col1{{8, 9, 10, 11, 12, 13}, {1, 0, 1, 1, 1, 1}};
  cudf::test::strings_column_wrapper col2(strings.begin(), strings.end());

  // creating a struct column will cause all child columns to be promoted to have validity
  cudf::test::structs_column_wrapper struct_col({col0, col1, col2}, struct_validity);

  // expect (1 offset (4 bytes) + (length of string if row is valid) + 1 validity bit) +
  //        (1 float + 1 validity bit) +
  //        (1 int16_t + 1 validity bit) +
  //        (1 validity bit)
  auto size_iter =
    cudf::detail::make_counting_transform_iterator(0, [&strings, &struct_validity](int i) {
      return (sizeof(float) * CHAR_BIT) + 1 + (sizeof(int16_t) * CHAR_BIT) + 1 +
             (static_cast<size_type>(strings[i].size()) * CHAR_BIT) +
             (sizeof(offset_type) * CHAR_BIT) + 1 + 1;
    });
  cudf::test::fixed_width_column_wrapper<size_type> expected_sizes(size_iter,
                                                                   size_iter + strings.size());

  return {struct_col.release(), expected_sizes.release()};
}

TEST_F(RowBitCount, StructsNoNulls)
{
  std::vector<std::string> strings{"abc", "daïs", "", "z", "bananas", "warp"};

  cudf::test::fixed_width_column_wrapper<float> col0{0, 1, 2, 3, 4, 5};
  cudf::test::fixed_width_column_wrapper<int16_t> col1{8, 9, 10, 11, 12, 13};
  cudf::test::strings_column_wrapper col2(strings.begin(), strings.end());

  cudf::test::structs_column_wrapper struct_col({col0, col1, col2});

  table_view t({struct_col});
  auto result = cudf::row_bit_count(t);

  // expect 1 offset (4 bytes) + (length of string) + 1 float + 1 int16_t
  auto size_iter = cudf::detail::make_counting_transform_iterator(0, [&strings](int i) {
    return ((sizeof(float) + sizeof(int16_t)) * CHAR_BIT) +
           ((static_cast<size_type>(strings[i].size()) + sizeof(offset_type)) * CHAR_BIT);
  });
  cudf::test::fixed_width_column_wrapper<size_type> expected(size_iter, size_iter + t.num_rows());

  CUDF_TEST_EXPECT_COLUMNS_EQUAL(expected, *result);
}

TEST_F(RowBitCount, StructsNulls)
{
  auto [struct_col, expected_sizes] = build_struct_column();
  table_view t({*struct_col});
  auto result = cudf::row_bit_count(t);

  CUDF_TEST_EXPECT_COLUMNS_EQUAL(*expected_sizes, *result);
}

TEST_F(RowBitCount, StructsNested)
{
  // struct<struct<int>, int16>
  cudf::test::fixed_width_column_wrapper<int> col0{0, 1, 2, 3, 4, 5};
  cudf::test::structs_column_wrapper inner_struct({col0});

  cudf::test::fixed_width_column_wrapper<int16_t> col1{8, 9, 10, 11, 12, 13};
  cudf::test::structs_column_wrapper struct_col({inner_struct, col1});

  table_view t({struct_col});
  auto result = cudf::row_bit_count(t);

  // expect num_rows * (4 + 2) bytes
  auto size_iter =
    cudf::detail::make_counting_transform_iterator(0, [&](int i) { return (4 + 2) * CHAR_BIT; });
  cudf::test::fixed_width_column_wrapper<size_type> expected(size_iter, size_iter + t.num_rows());
  CUDF_TEST_EXPECT_COLUMNS_EQUAL(expected, *result);
}

std::pair<std::unique_ptr<column>, std::unique_ptr<column>> build_nested_and_expected_column(
  std::vector<bool> const& struct_validity)
{
  // tests the "branching" case ->  list<struct<list> ...>>>

  // List<Struct<List<int>, float, int16>

  // Inner list column
  // clang-format off
  cudf::test::lists_column_wrapper<int> list{
    {1, 2, 3, 4, 5},
    {6, 7, 8},
    {33, 34, 35, 36, 37, 38, 39},
    {-1, -2},
    {-10, -11, -1, -20},
    {40, 41, 42},
    {100, 200, 300},
    {-100, -200, -300}};
  // clang-format on

  // floats
  std::vector<float> ages{5, 10, 15, 20, 4, 75, 16, -16};
  std::vector<bool> ages_validity = {1, 1, 1, 1, 0, 1, 0, 1};
  auto ages_column =
    cudf::test::fixed_width_column_wrapper<float>(ages.begin(), ages.end(), ages_validity.begin());

  // int16 values
  std::vector<int16_t> vals{-1, -2, -3, 1, 2, 3, 8, 9};
  auto i16_column = cudf::test::fixed_width_column_wrapper<int16_t>(vals.begin(), vals.end());

  // Assemble struct column
  auto struct_column =
    cudf::test::structs_column_wrapper({list, ages_column, i16_column}, struct_validity);

  // wrap in a list
  std::vector<int> outer_offsets{0, 1, 1, 3, 6, 7, 8};
  cudf::test::fixed_width_column_wrapper<int> outer_offsets_col(outer_offsets.begin(),
                                                                outer_offsets.end());
  auto const size = static_cast<column_view>(outer_offsets_col).size() - 1;

  cudf::test::fixed_width_column_wrapper<size_type> expected_sizes{276, 32, 520, 572, 212, 212};

  return {cudf::make_lists_column(static_cast<cudf::size_type>(size),
                                  outer_offsets_col.release(),
                                  struct_column.release(),
                                  cudf::UNKNOWN_NULL_COUNT,
                                  rmm::device_buffer{}),
          expected_sizes.release()};
}

std::unique_ptr<column> build_nested_column(std::vector<bool> const& struct_validity)
{
  // List<Struct<List<List<int>>, Struct<int16>>>

  // Inner list column
  // clang-format off
  cudf::test::lists_column_wrapper<int> list{
     {{1, 2, 3, 4, 5}, {2, 3}},
     {{6, 7, 8}, {8, 9}},
     {{1, 2}, {3, 4, 5}, {33, 34, 35, 36, 37, 38, 39}}};
  // clang-format on

  // Inner struct
  std::vector<int16_t> vals{-1, -2, -3};
  auto i16_column   = cudf::test::fixed_width_column_wrapper<int16_t>(vals.begin(), vals.end());
  auto inner_struct = cudf::test::structs_column_wrapper({i16_column});

  // outer struct
  auto outer_struct = cudf::test::structs_column_wrapper({list, inner_struct}, struct_validity);

  // wrap in a list
  std::vector<int> outer_offsets{0, 1, 1, 3};
  cudf::test::fixed_width_column_wrapper<int> outer_offsets_col(outer_offsets.begin(),
                                                                outer_offsets.end());
  auto const size = static_cast<column_view>(outer_offsets_col).size() - 1;
  return make_lists_column(static_cast<cudf::size_type>(size),
                           outer_offsets_col.release(),
                           outer_struct.release(),
                           cudf::UNKNOWN_NULL_COUNT,
                           rmm::device_buffer{});
}

TEST_F(RowBitCount, NestedTypes)
{
  // List<Struct<List<int>, float, List<int>, int16>
  {
    auto [col_no_nulls, expected_sizes] =
      build_nested_and_expected_column({1, 1, 1, 1, 1, 1, 1, 1});
    table_view no_nulls_t({*col_no_nulls});
    auto no_nulls_result = cudf::row_bit_count(no_nulls_t);

    auto col_nulls = build_nested_and_expected_column({0, 0, 1, 1, 1, 1, 1, 1}).first;
    table_view nulls_t({*col_nulls});
    auto nulls_result = cudf::row_bit_count(nulls_t);

    // List<Struct<List<int>, float, int16>
    //
    // this illustrates the difference between a row_bit_count
    // returning a pre-gather result, or a post-gather result.
    //
    // in a post-gather situation, the nulls in the struct would result in the values
    // nested in the list below to be dropped, resulting in smaller row sizes.
    //
    // however, for performance reasons, row_bit_count simply walks the data that is
    // currently there. so list rows that are null, but have a real span of
    // offsets (X, Y) instead of (X, X)  will end up getting the child data for those
    // rows included.
    //
    // if row_bit_count() is changed to return a post-gather result (which may be desirable),
    // the nulls_result case below will start failing and will need to be changed.
    //
    CUDF_TEST_EXPECT_COLUMNS_EQUAL(*expected_sizes, *no_nulls_result);
    CUDF_TEST_EXPECT_COLUMNS_EQUAL(*expected_sizes, *nulls_result);
  }

  // List<Struct<List<List<int>>, Struct<int16>>>
  {
    auto col_no_nulls = build_nested_column({1, 1, 1});
    table_view no_nulls_t({*col_no_nulls});
    auto no_nulls_result = cudf::row_bit_count(no_nulls_t);

    auto col_nulls = build_nested_column({1, 0, 1});
    table_view nulls_t({*col_nulls});
    auto nulls_result = cudf::row_bit_count(nulls_t);

    cudf::test::fixed_width_column_wrapper<size_type> expected_sizes{372, 32, 840};

    // same explanation as above
    CUDF_TEST_EXPECT_COLUMNS_EQUAL(expected_sizes, *no_nulls_result);
    CUDF_TEST_EXPECT_COLUMNS_EQUAL(expected_sizes, *nulls_result);
  }

  // test pushing/popping multiple times within one struct, and branch depth > 1
  //
  // Struct<int, List<int>, float, List<List<int16>>, Struct<List<int>, List<Struct<List<int>,
  // float>>, int8_t>>
  {
    cudf::test::lists_column_wrapper<int> l0{{1, 2, 3}, {4, 5}, {6, 7, 8, 9}, {5}};
    cudf::test::lists_column_wrapper<int16_t> l1{
      {{-1, -2}, {3, 4}}, {{4, 5}, {6, 7, 8}}, {{-6, -7}, {2}}, {{-11, -11}, {-12, -12}, {3}}};
    cudf::test::lists_column_wrapper<int> l2{{-1, -2}, {4, 5}, {-6, -7}, {1}};
    cudf::test::lists_column_wrapper<int> l3{{-1, -2, 0}, {5}, {-1, -6, -7}, {1, 2}};

    cudf::test::fixed_width_column_wrapper<int> c0{1, 2, 3, 4};
    cudf::test::fixed_width_column_wrapper<float> c1{1, 2, 3, 4};
    cudf::test::fixed_width_column_wrapper<int8_t> c2{1, 2, 3, 4};
    cudf::test::fixed_width_column_wrapper<float> c3{11, 12, 13, 14};

    // innermost List<Struct<List<int>>>
    auto innermost_struct = cudf::test::structs_column_wrapper({l3, c3});
    std::vector<int> l4_offsets{0, 1, 2, 3, 4};
    cudf::test::fixed_width_column_wrapper<int> l4_offsets_col(l4_offsets.begin(),
                                                               l4_offsets.end());
    auto const l4_size = l4_offsets.size() - 1;
    auto l4            = cudf::make_lists_column(static_cast<cudf::size_type>(l4_size),
                                      l4_offsets_col.release(),
                                      innermost_struct.release(),
                                      cudf::UNKNOWN_NULL_COUNT,
                                      rmm::device_buffer{});

    // inner struct
    std::vector<std::unique_ptr<column>> inner_struct_children;
    inner_struct_children.push_back(l2.release());
    inner_struct_children.push_back(std::move(l4));
    auto inner_struct = cudf::test::structs_column_wrapper(std::move(inner_struct_children));

    // outer struct
    auto struct_col = cudf::test::structs_column_wrapper({c0, l0, c1, l1, inner_struct, c2});

    table_view t({struct_col});
    auto result = cudf::row_bit_count(t);

    cudf::test::fixed_width_column_wrapper<size_type> expected_sizes{648, 568, 664, 568};

    CUDF_TEST_EXPECT_COLUMNS_EQUAL(expected_sizes, *result);
  }
}

TEST_F(RowBitCount, NullsInStringsList)
{
  using offsets_wrapper = cudf::test::fixed_width_column_wrapper<offset_type>;

  // clang-format off
  auto strings = std::vector<std::string>{ "daïs", "def", "", "z", "bananas", "warp", "", "zing" };
  auto valids  = std::vector<bool>{            1,     0,   0,  1,         0,      1,   1,     1 };
  // clang-format on

  cudf::test::strings_column_wrapper col(strings.begin(), strings.end(), valids.begin());

  auto offsets   = cudf::test::fixed_width_column_wrapper<int>{0, 2, 4, 6, 8};
  auto lists_col = cudf::make_lists_column(
    4,
    offsets_wrapper{0, 2, 4, 6, 8}.release(),
    cudf::test::strings_column_wrapper{strings.begin(), strings.end(), valids.begin()}.release(),
    0,
    {});
  CUDF_TEST_EXPECT_COLUMNS_EQUIVALENT(
    cudf::row_bit_count(table_view{{lists_col->view()}})->view(),
    cudf::test::fixed_width_column_wrapper<offset_type>{138, 106, 130, 130});
}

TEST_F(RowBitCount, EmptyChildColumnInListOfStrings)
{
  // Test with a list<string> column with 4 empty list rows.
  // Note: Since there are no strings in any of the lists,
  //       the lists column's child can be empty.
  auto offsets   = cudf::test::fixed_width_column_wrapper<offset_type>{0, 0, 0, 0, 0};
  auto lists_col = cudf::make_lists_column(
    4, offsets.release(), cudf::make_empty_column(cudf::data_type{cudf::type_id::STRING}), 0, {});

  CUDF_TEST_EXPECT_COLUMNS_EQUIVALENT(
    cudf::row_bit_count(table_view{{lists_col->view()}})->view(),
    cudf::test::fixed_width_column_wrapper<offset_type>{32, 32, 32, 32});
}

TEST_F(RowBitCount, EmptyChildColumnInListOfLists)
{
  // Test with a list<list> column with 4 empty list rows.
  // Note: Since there are no elements in any of the lists,
  //       the lists column's child can be empty.
  auto empty_child_lists_column = [] {
    auto exemplar = cudf::test::lists_column_wrapper<int32_t>{{0, 1, 2}, {3, 4, 5}};
    return cudf::empty_like(exemplar);
  };

  auto offsets   = cudf::test::fixed_width_column_wrapper<offset_type>{0, 0, 0, 0, 0};
  auto lists_col = cudf::make_lists_column(4, offsets.release(), empty_child_lists_column(), 0, {});

  CUDF_TEST_EXPECT_COLUMNS_EQUIVALENT(
    cudf::row_bit_count(table_view{{lists_col->view()}})->view(),
    cudf::test::fixed_width_column_wrapper<offset_type>{32, 32, 32, 32});
}

struct sum_functor {
  size_type const* s0;
  size_type const* s1;
  size_type const* s2;

  size_type operator() __device__(int i) { return s0[i] + s1[i] + s2[i]; }
};

TEST_F(RowBitCount, Table)
{
  // complex nested column
  auto [col0, col0_sizes] = build_nested_and_expected_column({1, 1, 1, 1, 1, 1, 1, 1});

  // struct column
  auto [col1, col1_sizes] = build_struct_column();

  // list column
  auto [col2, col2_sizes] = build_list_column<int16_t>();

  table_view t({*col0, *col1, *col2});
  auto result = cudf::row_bit_count(t);

  // sum all column sizes
  column_view cv0 = static_cast<column_view>(*col0_sizes);
  column_view cv1 = static_cast<column_view>(*col1_sizes);
  column_view cv2 = static_cast<column_view>(*col2_sizes);
  auto expected   = cudf::make_fixed_width_column(data_type{type_id::INT32}, t.num_rows());
  cudf::mutable_column_view mcv(*expected);
  thrust::transform(
    rmm::exec_policy(cudf::default_stream_value),
    thrust::make_counting_iterator(0),
    thrust::make_counting_iterator(0) + t.num_rows(),
    mcv.begin<size_type>(),
    sum_functor{cv0.data<size_type>(), cv1.data<size_type>(), cv2.data<size_type>()});
  CUDF_TEST_EXPECT_COLUMNS_EQUAL(*expected, *result);
}

TEST_F(RowBitCount, SlicedColumnsFixedWidth)
{
  auto const slice_size = 7;
  cudf::test::fixed_width_column_wrapper<int16_t> c0_unsliced{1, 2, 3, 4, 5, 6, 7, 8, 9, 10};
  auto c0 = cudf::slice(c0_unsliced, {2, 2 + slice_size});

  table_view t({c0});
  auto result = cudf::row_bit_count(t);

  cudf::test::fixed_width_column_wrapper<size_type> expected{16, 16, 16, 16, 16, 16, 16};

  CUDF_TEST_EXPECT_COLUMNS_EQUAL(expected, *result);
}

TEST_F(RowBitCount, SlicedColumnsStrings)
{
  auto const slice_size = 7;
  std::vector<std::string> strings{
    "banana", "metric", "imperial", "abc", "daïs", "", "fire", "def", "cudf", "xyzw"};
  cudf::test::strings_column_wrapper c0_unsliced(strings.begin(), strings.end());
  auto c0 = cudf::slice(c0_unsliced, {3, 3 + slice_size});

  table_view t({c0});
  auto result = cudf::row_bit_count(t);

  // expect 1 offset (4 bytes) + length of string per row
  auto size_iter = cudf::detail::make_counting_transform_iterator(0, [&strings](int i) {
    return (static_cast<size_type>(strings[i].size()) + sizeof(offset_type)) * CHAR_BIT;
  });
  cudf::test::fixed_width_column_wrapper<size_type> expected(size_iter + 3,
                                                             size_iter + 3 + slice_size);

  CUDF_TEST_EXPECT_COLUMNS_EQUAL(expected, *result);
}

TEST_F(RowBitCount, SlicedColumnsLists)
{
  auto const slice_size = 2;
  cudf::test::lists_column_wrapper<cudf::string_view> c0_unsliced{
    {{"banana", "v"}, {"cats"}},
    {{"dogs", "yay"}, {"xyz", ""}, {"daïs"}},
    {{"fast", "parrot"}, {"orange"}},
    {{"blue"}, {"red", "yellow"}, {"ultraviolet", "", "green"}}};
  auto c0 = cudf::slice(c0_unsliced, {1, 1 + slice_size});

  table_view t({c0});
  auto result = cudf::row_bit_count(t);

  cudf::test::fixed_width_column_wrapper<size_type> expected{408, 320};

  CUDF_TEST_EXPECT_COLUMNS_EQUAL(expected, *result);
}

TEST_F(RowBitCount, SlicedColumnsStructs)
{
  auto const slice_size = 7;

  cudf::test::fixed_width_column_wrapper<int16_t> c0{1, 2, 3, 4, 5, 6, 7, 8, 9, 10};
  std::vector<std::string> strings{
    "banana", "metric", "imperial", "abc", "daïs", "", "fire", "def", "cudf", "xyzw"};
  cudf::test::strings_column_wrapper c1(strings.begin(), strings.end());

  auto struct_col_unsliced = cudf::test::structs_column_wrapper({c0, c1});
  auto struct_col          = cudf::slice(struct_col_unsliced, {3, 3 + slice_size});

  table_view t({struct_col});
  auto result = cudf::row_bit_count(t);

  // expect 1 offset (4 bytes) + length of string per row + 1 int16_t per row
  auto size_iter = cudf::detail::make_counting_transform_iterator(0, [&strings](int i) {
    return (static_cast<size_type>(strings[i].size()) + sizeof(offset_type) + sizeof(int16_t)) *
           CHAR_BIT;
  });
  cudf::test::fixed_width_column_wrapper<size_type> expected(size_iter + 3,
                                                             size_iter + 3 + slice_size);

  CUDF_TEST_EXPECT_COLUMNS_EQUAL(expected, *result);
}

TEST_F(RowBitCount, EmptyTable)
{
  {
    cudf::table_view empty;
    auto result = cudf::row_bit_count(empty);
    CUDF_EXPECTS(result != nullptr && result->size() == 0, "Expected an empty column");
  }

  {
    auto strings = cudf::make_empty_column(type_id::STRING);
    auto ints    = cudf::make_empty_column(type_id::INT32);
    cudf::table_view empty({*strings, *ints});

    auto result = cudf::row_bit_count(empty);
    CUDF_EXPECTS(result != nullptr && result->size() == 0, "Expected an empty column");
  }
}
