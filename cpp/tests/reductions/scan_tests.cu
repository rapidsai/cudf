#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <cstdlib>
#include <iostream>
#include <vector>
#include <algorithm>
#include <iterator>
#include <type_traits>

#include <tests/utilities/base_fixture.hpp>
#include <tests/utilities/type_lists.hpp>
//TODO remove after PR 3490 merge
#include <tests/utilities/legacy/cudf_test_utils.cuh>
#include <tests/utilities/column_wrapper.hpp>
#include <tests/utilities/column_utilities.hpp>

#include <cudf/cudf.h>
#include <cudf/reduction.hpp>

#include <thrust/device_vector.h>

#include <cudf/detail/aggregation/aggregation.hpp>
using aggregation = cudf::experimental::aggregation;
using cudf::experimental::scan_type;
using cudf::experimental::include_nulls;
using cudf::column_view;

void print_view(column_view const& view, const char* msg = nullptr) {
    std::cout << msg<< " {";
    cudf::test::print(view); std::cout << "}\n";
}

// This is the main test feature
template <typename T>
struct ScanTest : public cudf::test::BaseFixture
{
    void scan_test(
        cudf::test::fixed_width_column_wrapper<T> const col_in,
        cudf::test::fixed_width_column_wrapper<T> const expected_col_out,
        std::unique_ptr<aggregation> const &agg, scan_type inclusive)
    {
        bool do_print = false;

        auto int_values = cudf::test::to_host<T>(col_in);
        auto exact_values = cudf::test::to_host<T>(expected_col_out);
        this->val_check(std::get<0>(int_values), do_print, "input = ");
        this->val_check(std::get<0>(exact_values), do_print, "exact = ");

        const column_view input_view = col_in;
        std::unique_ptr<cudf::column> col_out;

        CUDF_EXPECT_NO_THROW( col_out = cudf::experimental::scan(input_view, agg, inclusive) );
        const column_view result_view = col_out->view();

        cudf::test::expect_column_properties_equal(input_view, result_view);
        cudf::test::expect_columns_equal(expected_col_out, result_view);

        auto host_result = cudf::test::to_host<T>(result_view);
        this->val_check(std::get<0>(host_result), do_print, "result = ");
    }

    template <typename Ti>
    void val_check(std::vector<Ti> const & v, bool do_print=false, const char* msg = nullptr){
        if( do_print ){
            std::cout << msg << " {";
            std::for_each(v.begin(), v.end(), [](Ti i){ std::cout << ", " <<  i;});
            std::cout << "}"  << std::endl;
        }
        range_check(v);
    }

    // make sure all elements in the range of sint8([-128, 127])
    template <typename Ti>
    void range_check(std::vector<Ti> const & v){
        std::for_each(v.begin(), v.end(),
            [](Ti i){
                ASSERT_GE(static_cast<int>(i), -128);
                ASSERT_LT(static_cast<int>(i),  128);
            });
    }
};

using Types = cudf::test::NumericTypes;
//using Types = testing::Types<int32_t>;

TYPED_TEST_CASE(ScanTest, Types);

// ------------------------------------------------------------------------
TYPED_TEST(ScanTest, Min)
{
    std::vector<TypeParam>  v({123, 64, 63, 99, -5, 123, -16, -120, -111});
    std::vector<bool> b({  1,  0,  1,  1,  1,   1,   0,    1,    1});
    std::vector<TypeParam> exact(v.size());

    std::transform(v.cbegin(), v.cend(),
        exact.begin(),
        [acc=v[0]](auto i) mutable { acc = std::min(acc, i); return acc; }
        );

    this->scan_test({v.begin(), v.end()}, 
                    {exact.begin(), exact.end()},
                    cudf::experimental::make_min_aggregation(), scan_type::INCLUSIVE);

    std::transform(v.cbegin(), v.cend(), b.begin(),
        exact.begin(),
        [acc=v[0]](auto i, bool b) mutable { if(b) acc = std::min(acc, i); return acc; }
        );

    this->scan_test({v.begin(), v.end(), b.begin()}, 
                    {exact.begin(), exact.end(), b.begin()},
                    cudf::experimental::make_min_aggregation(), scan_type::INCLUSIVE);
}

TYPED_TEST(ScanTest, Max)
{
    std::vector<TypeParam>  v({-120, 5, 0, -120, -111, 64, 63, 99, 123, -16});
    std::vector<bool> b({   1, 0, 1,    1,    1,  1,  0,  1,   1,   1});
    std::vector<TypeParam> exact(v.size());

    std::transform(v.cbegin(), v.cend(),
        exact.begin(),
        [acc=v[0]](auto i) mutable { acc = std::max(acc, i); return acc; }
        );

    this->scan_test({v.begin(), v.end()}, 
                    {exact.begin(), exact.end()},
                    cudf::experimental::make_max_aggregation(), scan_type::INCLUSIVE);

    std::transform(v.cbegin(), v.cend(), b.begin(),
        exact.begin(),
        [acc=v[0]](auto i, bool b) mutable { if(b) acc = std::max(acc, i); return acc; }
        );

    this->scan_test({v.begin(), v.end(), b.begin()}, 
                    {exact.begin(), exact.end(), b.begin()},
                    cudf::experimental::make_max_aggregation(), scan_type::INCLUSIVE);
}


TYPED_TEST(ScanTest, Product)
{
    std::vector<TypeParam>  v({5, -1, 1, 3, -2, 4});
    std::vector<bool> b({1,  1, 1, 0,  1, 1});
    std::vector<TypeParam> exact(v.size());

    std::transform(v.cbegin(), v.cend(),
        exact.begin(),
        [acc=1](auto i) mutable { acc *= i; return acc; }
        );

    this->scan_test({v.begin(), v.end()}, 
                    {exact.begin(), exact.end()},
                    cudf::experimental::make_product_aggregation(), scan_type::INCLUSIVE);

    std::transform(v.cbegin(), v.cend(), b.begin(),
        exact.begin(),
        [acc=1](auto i, bool b) mutable { if(b) acc *= i; return acc; }
        );

    this->scan_test({v.begin(), v.end(), b.begin()}, 
                    {exact.begin(), exact.end(), b.begin()},
                    cudf::experimental::make_product_aggregation(), scan_type::INCLUSIVE);
}

TYPED_TEST(ScanTest, Sum)
{
    std::vector<TypeParam>  v({-120, 5, 6, 113, -111, 64, -63, 9, 34, -16});
    std::vector<bool> b({   1, 0, 1,   1,    0,  0,   1, 1,  1,   1});
    std::vector<TypeParam> exact(v.size());

    std::transform(v.cbegin(), v.cend(),
        exact.begin(),
        [acc=0](auto i) mutable { acc += i; return acc; }
        );

    this->scan_test({v.begin(), v.end()}, 
                    {exact.begin(), exact.end()},
                    cudf::experimental::make_sum_aggregation(), scan_type::INCLUSIVE);

    std::transform(v.cbegin(), v.cend(), b.begin(),
        exact.begin(),
        [acc=0](auto i, bool b) mutable { if(b) acc += i; return acc; }
        );

    this->scan_test({v.begin(), v.end(), b.begin()}, 
                    {exact.begin(), exact.end(), b.begin()},
                    cudf::experimental::make_sum_aggregation(), scan_type::INCLUSIVE);
}

struct ScanStringTest : public cudf::test::BaseFixture {
  void scan_test(cudf::test::strings_column_wrapper const& col_in,
                 cudf::test::strings_column_wrapper const& expected_col_out,
                 std::unique_ptr<aggregation> const &agg, scan_type inclusive) 
  {
    bool do_print = false;
    if (do_print) {
      std::cout << "input = {";  cudf::test::print(col_in);  std::cout<<"}\n";
      std::cout << "expect = {";  cudf::test::print(expected_col_out);  std::cout<<"}\n";
    }

    const column_view input_view = col_in;
    std::unique_ptr<cudf::column> col_out;

    CUDF_EXPECT_NO_THROW(col_out = cudf::experimental::scan(input_view, agg, inclusive));
    const column_view result_view = col_out->view();

    cudf::test::expect_column_properties_equal(input_view, result_view);
    cudf::test::expect_columns_equal(expected_col_out, result_view);

    if (do_print) {
      std::cout << "result = {"; cudf::test::print(result_view); std::cout<<"}\n";
    }
  }
};

TEST_F(ScanStringTest, Min)
{
  // data and valid arrays
  std::vector<std::string> v({"one", "two", "three", "four", "five", "six", "seven", "eight", "nine"});
  std::vector<bool>        b({    1,     0,       1,      1,      0,     0,       1,       1,      1});
  std::vector<std::string> exact(v.size());

  std::transform(v.cbegin(), v.cend(),
                 exact.begin(),
                 [acc = v[0]](auto i) mutable { acc = std::min(acc, i); return acc; });

  // string column without nulls
  cudf::test::strings_column_wrapper col_nonulls(v.begin(), v.end());
  cudf::test::strings_column_wrapper expected1(exact.begin(), exact.end());
  this->scan_test(col_nonulls, expected1,
                  cudf::experimental::make_min_aggregation(), scan_type::INCLUSIVE);

  std::transform(v.cbegin(), v.cend(), b.begin(),
        exact.begin(),
        [acc=v[0]](auto i, bool b) mutable { if(b) acc = std::min(acc, i); return acc; }
        );
  // string column with nulls
  cudf::test::strings_column_wrapper col_nulls(v.begin(), v.end(), b.begin());
  cudf::test::strings_column_wrapper expected2(exact.begin(), exact.end(), b.begin());
  this->scan_test(col_nulls, expected2,
                  cudf::experimental::make_min_aggregation(), scan_type::INCLUSIVE);
}

TEST_F(ScanStringTest, Max)
{
  // data and valid arrays
  std::vector<std::string> v({"one", "two", "three", "four", "five", "six", "seven", "eight", "nine"});
  std::vector<bool>        b({    1,     0,       1,      1,      0,     0,       1,       1,      1});
  std::vector<std::string> exact(v.size());

  std::transform(v.cbegin(), v.cend(),
                 exact.begin(),
                 [acc = v[0]](auto i) mutable { acc = std::max(acc, i); return acc; });

  // string column without nulls
  cudf::test::strings_column_wrapper col_nonulls(v.begin(), v.end());
  cudf::test::strings_column_wrapper expected1(exact.begin(), exact.end());
  this->scan_test(col_nonulls, expected1, cudf::experimental::make_max_aggregation(), scan_type::INCLUSIVE);

  std::transform(v.cbegin(), v.cend(), b.begin(),
        exact.begin(),
        [acc=v[0]](auto i, bool b) mutable { if(b) acc = std::max(acc, i); return acc; }
        );
  // string column with nulls
  cudf::test::strings_column_wrapper col_nulls(v.begin(), v.end(), b.begin());
  cudf::test::strings_column_wrapper expected2(exact.begin(), exact.end(), b.begin());
  this->scan_test(col_nulls, expected2, cudf::experimental::make_max_aggregation(), scan_type::INCLUSIVE);
}

TYPED_TEST(ScanTest, skip_nulls)
{
  bool do_print=false;
  std::vector<TypeParam> v{1,2,3,4,5,6,7,8,1,1};
  std::vector<bool>      b{1,1,1,1,1,0,1,0,1,1};
  cudf::test::fixed_width_column_wrapper<TypeParam> const col_in{v.begin(), v.end(),
                                                            b.begin()};
  const column_view input_view = col_in;
  std::unique_ptr<cudf::column> col_out;
  
  //test output calculation
  std::vector<TypeParam> out_v(input_view.size());
  std::vector<bool>      out_b(input_view.size());

  std::transform(v.cbegin(), v.cend(), b.cbegin(),
      out_v.begin(),
      [acc=0](auto i, bool b) mutable { if(b) (acc += i); return acc; }
      );
  std::transform(b.cbegin(), b.cend(),
      out_b.begin(),
      [acc=true](auto i) mutable { acc = acc && i; return acc; }
      );

  //skipna=true (default)
  CUDF_EXPECT_NO_THROW(col_out = cudf::experimental::scan(input_view, 
                      cudf::experimental::make_sum_aggregation(), scan_type::INCLUSIVE, include_nulls::NO));
  cudf::test::fixed_width_column_wrapper<TypeParam> expected_col_out1{
      out_v.begin(), out_v.end(), b.cbegin()};
  cudf::test::expect_column_properties_equal(expected_col_out1, col_out->view());
  cudf::test::expect_columns_equal(expected_col_out1, col_out->view());
  if(do_print) {
    print_view(expected_col_out1, "expect = ");
    print_view(col_out->view(),   "result = ");
  }

  //skipna=false
  CUDF_EXPECT_NO_THROW(col_out = cudf::experimental::scan(input_view, 
                      cudf::experimental::make_sum_aggregation(), scan_type::INCLUSIVE, include_nulls::YES));
  cudf::test::fixed_width_column_wrapper<TypeParam> expected_col_out2{
      out_v.begin(), out_v.end(), out_b.begin()};
  if(do_print) {
    print_view(expected_col_out2, "expect = ");
    print_view(col_out->view(),   "result = ");
  }
  cudf::test::expect_column_properties_equal(expected_col_out2, col_out->view());
  cudf::test::expect_columns_equal(expected_col_out2, col_out->view());
}

TEST_F(ScanStringTest, skip_nulls)
{
  bool do_print=false;
  // data and valid arrays
  std::vector<std::string> v({"one", "two", "three", "four", "five", "six", "seven", "eight", "nine"});
  std::vector<bool>        b({    1,     1,       1,      0,      0,     0,       1,       1,      1});
  std::vector<std::string> exact(v.size());
  std::vector<bool>      out_b(v.size());

  // test output calculation
  std::transform(v.cbegin(), v.cend(), b.begin(),
        exact.begin(),
        [acc=v[0]](auto i, bool b) mutable { if(b) acc = std::max(acc, i); return acc; }
        );
  std::transform(b.cbegin(), b.cend(),
      out_b.begin(),
      [acc=true](auto i) mutable { acc = acc && i; return acc; }
      );
  // string column with nulls
  cudf::test::strings_column_wrapper col_nulls(v.begin(), v.end(), b.begin());
  cudf::test::strings_column_wrapper expected2(exact.begin(), exact.end(), out_b.begin());
  std::unique_ptr<cudf::column> col_out;
  //skipna=false
  CUDF_EXPECT_NO_THROW(col_out = cudf::experimental::scan(col_nulls, 
    cudf::experimental::make_max_aggregation(), scan_type::INCLUSIVE, include_nulls::YES));
  if(do_print) {
    print_view(expected2, "expect = ");
    print_view(col_out->view(),   "result = ");
  }
  cudf::test::expect_column_properties_equal(expected2, col_out->view());
  cudf::test::expect_columns_equal(expected2, col_out->view());

  //Exclusive scan string not supported.
  CUDF_EXPECT_THROW_MESSAGE((cudf::experimental::scan(col_nulls, 
  cudf::experimental::make_min_aggregation(), scan_type::EXCLUSIVE, include_nulls::NO)),
  "String types supports only inclusive min/max for `cudf::scan`");

  CUDF_EXPECT_THROW_MESSAGE((cudf::experimental::scan(col_nulls, 
  cudf::experimental::make_min_aggregation(), scan_type::EXCLUSIVE, include_nulls::YES)),
  "String types supports only inclusive min/max for `cudf::scan`");
}

TYPED_TEST(ScanTest, EmptyColumnskip_nulls)
{
  bool do_print=false;
  std::vector<TypeParam> v{};
  std::vector<bool>      b{};
  cudf::test::fixed_width_column_wrapper<TypeParam> const col_in{v.begin(), v.end(),
                                                            b.begin()};
  std::unique_ptr<cudf::column> col_out;
  
  //test output calculation
  std::vector<TypeParam> out_v(v.size());
  std::vector<bool>      out_b(v.size());
  
  //skipna=true (default)
  CUDF_EXPECT_NO_THROW(col_out = cudf::experimental::scan(col_in, 
    cudf::experimental::make_sum_aggregation(), scan_type::INCLUSIVE, include_nulls::NO));
  cudf::test::fixed_width_column_wrapper<TypeParam> expected_col_out1{
      out_v.begin(), out_v.end(), b.cbegin()};
  cudf::test::expect_column_properties_equal(expected_col_out1, col_out->view());
  cudf::test::expect_columns_equal(expected_col_out1, col_out->view());
  if(do_print) {
    print_view(expected_col_out1, "expect = ");
    print_view(col_out->view(),   "result = ");
  }

  //skipna=false
  CUDF_EXPECT_NO_THROW(col_out = cudf::experimental::scan(col_in, 
  cudf::experimental::make_sum_aggregation(), scan_type::INCLUSIVE, include_nulls::YES));
  cudf::test::fixed_width_column_wrapper<TypeParam> expected_col_out2{
      out_v.begin(), out_v.end(), out_b.begin()};
  if(do_print) {
    print_view(expected_col_out2, "expect = ");
    print_view(col_out->view(),   "result = ");
  }
  cudf::test::expect_column_properties_equal(expected_col_out2, col_out->view());
  cudf::test::expect_columns_equal(expected_col_out2, col_out->view());
}