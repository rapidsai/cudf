#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <cstdlib>
#include <iostream>
#include <vector>
#include <algorithm>
#include <iterator>
#include <type_traits>

#include <gtest/gtest.h>

#include <cudf/cudf.h>
#include <cudf/reduction.hpp>

#include <thrust/device_vector.h>

#include <tests/utilities/legacy/cudf_test_fixtures.h>
#include <tests/utilities/legacy/cudf_test_utils.cuh>
#include <tests/utilities/legacy/column_wrapper.cuh>

// This is the main test feature
template <typename T>
struct ScanTest : public GdfTest
{
    void scan_test(std::vector<int> const & int_values,
        std::vector<int> const & exact_values,
        gdf_scan_op op, bool inclusive)
    {
        bool do_print = false;

        this->val_check(int_values, do_print, "input = ");
        this->val_check(exact_values, do_print, "exact = ");

        cudf::size_type col_size = int_values.size();
        std::vector<T> input_values(col_size);

        std::transform(int_values.begin(), int_values.end(),
            input_values.begin(),
           [](int x) { T t(x) ; return t; } );

        cudf::test::column_wrapper<T> const col_in(input_values);
        const gdf_column * raw_input = col_in.get();

        cudf::test::column_wrapper<T> col_out(col_size);
        gdf_column * raw_output = col_out.get();

        CUDF_EXPECT_NO_THROW( cudf::scan(raw_input, raw_output, op, inclusive) );

        using UnderlyingType = T;
        auto tuple_host_result = col_out.to_host();
        auto host_result = std::get<0>(tuple_host_result);

        this->val_check(host_result, do_print, "result = ");

        std::equal(exact_values.begin(), exact_values.end(),
            host_result.begin(), host_result.end(),
            [](int x, UnderlyingType y) {
                EXPECT_EQ(UnderlyingType(x), y); return true; });
    }

    template <typename Ti>
    void val_check(std::vector<Ti> const & v, bool do_print=false, const char* msg = nullptr){
        if( do_print ){
            std::cout << msg << " {";
            std::for_each(v.begin(), v.end(), [](Ti i){ std::cout << ", " <<  i;});
            std::cout << "}"  << std::endl;
        }
        range_check(v);
    }

    // make sure all elements in the range of sint8([-128, 127])
    template <typename Ti>
    void range_check(std::vector<Ti> const & v){
        std::for_each(v.begin(), v.end(),
            [](Ti i){
                ASSERT_GE(static_cast<int>(i), -128);
                ASSERT_LT(static_cast<int>(i),  128);
            });
    }

};

using Types = testing::Types<
    int8_t,int16_t, int32_t, int64_t, float, double, cudf::bool8>;

TYPED_TEST_CASE(ScanTest, Types);

// ------------------------------------------------------------------------
TYPED_TEST(ScanTest, Min)
{
    std::vector<int> v({123, 64, 63, 99, -5, 123, -16, -120, -111});
    std::vector<int> exact;
    int acc(v[0]);

    std::for_each(v.begin(), v.end(),
        [&acc, &exact](int i){
            acc = std::min(acc, i); exact.push_back(acc);
        }
    );

    this->scan_test(v, exact, GDF_SCAN_MIN, true);
}

TYPED_TEST(ScanTest, Max)
{
    std::vector<int> v({-120, 5, 0, -120, -111, 64, 63, 99, 123, -16});

    std::vector<int> exact;
    int acc(v[0]);

    std::for_each(v.begin(), v.end(),
        [&acc, &exact](int i){
            acc = std::max(acc, i); exact.push_back(acc);
        }
    );

    this->scan_test(v, exact, GDF_SCAN_MAX, true);
}


TYPED_TEST(ScanTest, Product)
{
    std::vector<int> v({5, -1, 1, 3, -2, 4});

    std::vector<int> exact;
    int acc(1);
    std::for_each(v.begin(), v.end(),
        [&acc, &exact](int i){ acc *= i; exact.push_back(acc); });

    this->scan_test(v, exact, GDF_SCAN_PRODUCT, true);
}

TYPED_TEST(ScanTest, Sum)
{
    std::vector<int> v({-120, 5, 6, 113, -111, 64, -63, 9, 34, -16});

    std::vector<int> exact;
    int acc(0);
    std::for_each(v.begin(), v.end(),
        [&acc, &exact](int i){ acc += i; exact.push_back(acc); });

    this->scan_test(v, exact, GDF_SCAN_SUM, true);
}


