/*
 * Copyright (c) 2022-2023, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "row_operator_tests_utilities.hpp"

#include <cudf/column/column_factories.hpp>
#include <cudf/column/column_view.hpp>
#include <cudf/table/experimental/row_operators.cuh>

#include <rmm/cuda_stream_view.hpp>
#include <rmm/exec_policy.hpp>

#include <thrust/iterator/counting_iterator.h>
#include <thrust/sequence.h>
#include <thrust/sort.h>
#include <thrust/transform.h>

template <typename PhysicalElementComparator>
std::unique_ptr<cudf::column> self_comparison(cudf::table_view input,
                                              std::vector<cudf::order> const& column_order,
                                              PhysicalElementComparator comparator)
{
  rmm::cuda_stream_view stream{cudf::get_default_stream()};

  auto const table_comparator =
    cudf::experimental::row::lexicographic::self_comparator{input, column_order, {}, stream};

  auto output = cudf::make_numeric_column(
    cudf::data_type(cudf::type_id::BOOL8), input.num_rows(), cudf::mask_state::UNALLOCATED);

  if (cudf::detail::has_nested_columns(input)) {
    thrust::transform(rmm::exec_policy(stream),
                      thrust::make_counting_iterator(0),
                      thrust::make_counting_iterator(input.num_rows()),
                      thrust::make_counting_iterator(0),
                      output->mutable_view().data<bool>(),
                      table_comparator.less<true>(cudf::nullate::NO{}, comparator));
  } else {
    thrust::transform(rmm::exec_policy(stream),
                      thrust::make_counting_iterator(0),
                      thrust::make_counting_iterator(input.num_rows()),
                      thrust::make_counting_iterator(0),
                      output->mutable_view().data<bool>(),
                      table_comparator.less<false>(cudf::nullate::NO{}, comparator));
  }
  return output;
}

using physical_comparator_t = cudf::experimental::row::lexicographic::physical_element_comparator;
using sorting_comparator_t =
  cudf::experimental::row::lexicographic::sorting_physical_element_comparator;

template std::unique_ptr<cudf::column> self_comparison<physical_comparator_t>(
  cudf::table_view input,
  std::vector<cudf::order> const& column_order,
  physical_comparator_t comparator);
template std::unique_ptr<cudf::column> self_comparison<sorting_comparator_t>(
  cudf::table_view input,
  std::vector<cudf::order> const& column_order,
  sorting_comparator_t comparator);

template <typename PhysicalElementComparator>
std::unique_ptr<cudf::column> two_table_comparison(cudf::table_view lhs,
                                                   cudf::table_view rhs,
                                                   std::vector<cudf::order> const& column_order,
                                                   PhysicalElementComparator comparator)
{
  rmm::cuda_stream_view stream{cudf::get_default_stream()};

  auto const table_comparator = cudf::experimental::row::lexicographic::two_table_comparator{
    lhs, rhs, column_order, {}, stream};
  auto const lhs_it = cudf::experimental::row::lhs_iterator(0);
  auto const rhs_it = cudf::experimental::row::rhs_iterator(0);

  auto output = cudf::make_numeric_column(
    cudf::data_type(cudf::type_id::BOOL8), lhs.num_rows(), cudf::mask_state::UNALLOCATED);

  if (cudf::detail::has_nested_columns(lhs) || cudf::detail::has_nested_columns(rhs)) {
    thrust::transform(rmm::exec_policy(stream),
                      lhs_it,
                      lhs_it + lhs.num_rows(),
                      rhs_it,
                      output->mutable_view().data<bool>(),
                      table_comparator.less<true>(cudf::nullate::NO{}, comparator));
  } else {
    thrust::transform(rmm::exec_policy(stream),
                      lhs_it,
                      lhs_it + lhs.num_rows(),
                      rhs_it,
                      output->mutable_view().data<bool>(),
                      table_comparator.less<false>(cudf::nullate::NO{}, comparator));
  }
  return output;
}

template std::unique_ptr<cudf::column> two_table_comparison<physical_comparator_t>(
  cudf::table_view lhs,
  cudf::table_view rhs,
  std::vector<cudf::order> const& column_order,
  physical_comparator_t comparator);
template std::unique_ptr<cudf::column> two_table_comparison<sorting_comparator_t>(
  cudf::table_view lhs,
  cudf::table_view rhs,
  std::vector<cudf::order> const& column_order,
  sorting_comparator_t comparator);

template <typename PhysicalElementComparator>
std::unique_ptr<cudf::column> sorted_order(
  std::shared_ptr<cudf::experimental::row::lexicographic::preprocessed_table> preprocessed_input,
  cudf::size_type num_rows,
  bool has_nested,
  PhysicalElementComparator comparator,
  rmm::cuda_stream_view stream)
{
  auto output = cudf::make_numeric_column(cudf::data_type(cudf::type_to_id<cudf::size_type>()),
                                          num_rows,
                                          cudf::mask_state::UNALLOCATED,
                                          stream);
  auto const out_begin = output->mutable_view().begin<cudf::size_type>();
  thrust::sequence(rmm::exec_policy(stream), out_begin, out_begin + num_rows, 0);

  auto const table_comparator =
    cudf::experimental::row::lexicographic::self_comparator{preprocessed_input};
  if (has_nested) {
    auto const comp = table_comparator.less<true>(cudf::nullate::NO{}, comparator);
    thrust::stable_sort(rmm::exec_policy(stream), out_begin, out_begin + num_rows, comp);
  } else {
    auto const comp = table_comparator.less<false>(cudf::nullate::NO{}, comparator);
    thrust::stable_sort(rmm::exec_policy(stream), out_begin, out_begin + num_rows, comp);
  }

  return output;
}

template std::unique_ptr<cudf::column> sorted_order<physical_comparator_t>(
  std::shared_ptr<cudf::experimental::row::lexicographic::preprocessed_table> preprocessed_input,
  cudf::size_type num_rows,
  bool has_nested,
  physical_comparator_t comparator,
  rmm::cuda_stream_view stream);
template std::unique_ptr<cudf::column> sorted_order<sorting_comparator_t>(
  std::shared_ptr<cudf::experimental::row::lexicographic::preprocessed_table> preprocessed_input,
  cudf::size_type num_rows,
  bool has_nested,
  sorting_comparator_t comparator,
  rmm::cuda_stream_view stream);

template <typename PhysicalElementComparator>
std::unique_ptr<cudf::column> two_table_equality(cudf::table_view lhs,
                                                 cudf::table_view rhs,
                                                 std::vector<cudf::order> const& column_order,
                                                 PhysicalElementComparator comparator)
{
  rmm::cuda_stream_view stream{cudf::get_default_stream()};

  auto const table_comparator =
    cudf::experimental::row::equality::two_table_comparator{lhs, rhs, stream};

  auto const lhs_it = cudf::experimental::row::lhs_iterator(0);
  auto const rhs_it = cudf::experimental::row::rhs_iterator(0);

  auto output = cudf::make_numeric_column(
    cudf::data_type(cudf::type_id::BOOL8), lhs.num_rows(), cudf::mask_state::UNALLOCATED);

  if (cudf::detail::has_nested_columns(lhs) or cudf::detail::has_nested_columns(rhs)) {
    auto const equal_comparator =
      table_comparator.equal_to<true>(cudf::nullate::NO{}, cudf::null_equality::EQUAL, comparator);

    thrust::transform(rmm::exec_policy(stream),
                      lhs_it,
                      lhs_it + lhs.num_rows(),
                      rhs_it,
                      output->mutable_view().data<bool>(),
                      equal_comparator);
  } else {
    auto const equal_comparator =
      table_comparator.equal_to<false>(cudf::nullate::NO{}, cudf::null_equality::EQUAL, comparator);

    thrust::transform(rmm::exec_policy(stream),
                      lhs_it,
                      lhs_it + lhs.num_rows(),
                      rhs_it,
                      output->mutable_view().data<bool>(),
                      equal_comparator);
  }
  return output;
}

using physical_equality_t = cudf::experimental::row::equality::physical_equality_comparator;
using nan_equality_t = cudf::experimental::row::equality::nan_equal_physical_equality_comparator;

template std::unique_ptr<cudf::column> two_table_equality<physical_equality_t>(
  cudf::table_view lhs,
  cudf::table_view rhs,
  std::vector<cudf::order> const& column_order,
  physical_equality_t comparator);
template std::unique_ptr<cudf::column> two_table_equality<nan_equality_t>(
  cudf::table_view lhs,
  cudf::table_view rhs,
  std::vector<cudf::order> const& column_order,
  nan_equality_t comparator);
