/*
 * Copyright (c) 2022, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <cudf_test/base_fixture.hpp>
#include <cudf_test/column_utilities.hpp>
#include <cudf_test/column_wrapper.hpp>
#include <cudf_test/type_lists.hpp>

#include <cudf/column/column_view.hpp>
#include <cudf/table/experimental/row_operators.cuh>
#include <cudf/table/row_operators.cuh>
#include <cudf/table/table_view.hpp>
#include <cudf/utilities/default_stream.hpp>

#include <rmm/cuda_stream_view.hpp>
#include <rmm/exec_policy.hpp>

#include <thrust/iterator/counting_iterator.h>
#include <thrust/transform.h>

#include <cmath>
#include <vector>

template <typename T>
struct TypedTableViewTest : public cudf::test::BaseFixture {
};

using NumericTypesNotBool =
  cudf::test::Concat<cudf::test::IntegralTypesNotBool, cudf::test::FloatingPointTypes>;
TYPED_TEST_SUITE(TypedTableViewTest, NumericTypesNotBool);

template <typename PhysicalElementComparator>
auto self_comparison(cudf::table_view input,
                     std::vector<cudf::order> const& column_order,
                     PhysicalElementComparator comparator)
{
  rmm::cuda_stream_view stream{cudf::get_default_stream()};

  auto const table_comparator =
    cudf::experimental::row::lexicographic::self_comparator{input, column_order, {}, stream};

  auto output = cudf::make_numeric_column(
    cudf::data_type(cudf::type_id::BOOL8), input.num_rows(), cudf::mask_state::UNALLOCATED);

  if (cudf::detail::has_nested_columns(input)) {
    thrust::transform(rmm::exec_policy(stream),
                      thrust::make_counting_iterator(0),
                      thrust::make_counting_iterator(input.num_rows()),
                      thrust::make_counting_iterator(0),
                      output->mutable_view().data<bool>(),
                      table_comparator.less<true>(cudf::nullate::NO{}, comparator));
  } else {
    thrust::transform(rmm::exec_policy(stream),
                      thrust::make_counting_iterator(0),
                      thrust::make_counting_iterator(input.num_rows()),
                      thrust::make_counting_iterator(0),
                      output->mutable_view().data<bool>(),
                      table_comparator.less<false>(cudf::nullate::NO{}, comparator));
  }
  return output;
}

template <typename PhysicalElementComparator>
auto two_table_comparison(cudf::table_view lhs,
                          cudf::table_view rhs,
                          std::vector<cudf::order> const& column_order,
                          PhysicalElementComparator comparator)
{
  rmm::cuda_stream_view stream{cudf::get_default_stream()};

  auto const table_comparator = cudf::experimental::row::lexicographic::two_table_comparator{
    lhs, rhs, column_order, {}, stream};
  auto const lhs_it = cudf::experimental::row::lhs_iterator(0);
  auto const rhs_it = cudf::experimental::row::rhs_iterator(0);

  auto output = cudf::make_numeric_column(
    cudf::data_type(cudf::type_id::BOOL8), lhs.num_rows(), cudf::mask_state::UNALLOCATED);

  if (cudf::detail::has_nested_columns(lhs) || cudf::detail::has_nested_columns(rhs)) {
    thrust::transform(rmm::exec_policy(stream),
                      lhs_it,
                      lhs_it + lhs.num_rows(),
                      rhs_it,
                      output->mutable_view().data<bool>(),
                      table_comparator.less<true>(cudf::nullate::NO{}, comparator));
  } else {
    thrust::transform(rmm::exec_policy(stream),
                      lhs_it,
                      lhs_it + lhs.num_rows(),
                      rhs_it,
                      output->mutable_view().data<bool>(),
                      table_comparator.less<false>(cudf::nullate::NO{}, comparator));
  }
  return output;
}

template <typename PhysicalElementComparator>
auto self_equality(cudf::table_view input,
                   std::vector<cudf::order> const& column_order,
                   PhysicalElementComparator comparator)
{
  rmm::cuda_stream_view stream{cudf::get_default_stream()};

  auto const table_comparator = cudf::experimental::row::equality::self_comparator{input, stream};
  auto const equal_comparator =
    table_comparator.equal_to(cudf::nullate::NO{}, cudf::null_equality::EQUAL, comparator);

  auto output = cudf::make_numeric_column(
    cudf::data_type(cudf::type_id::BOOL8), input.num_rows(), cudf::mask_state::UNALLOCATED);

  thrust::transform(rmm::exec_policy(stream),
                    thrust::make_counting_iterator(0),
                    thrust::make_counting_iterator(input.num_rows()),
                    thrust::make_counting_iterator(0),
                    output->mutable_view().data<bool>(),
                    equal_comparator);
  return output;
}

template <typename PhysicalElementComparator>
auto two_table_equality(cudf::table_view lhs,
                        cudf::table_view rhs,
                        std::vector<cudf::order> const& column_order,
                        PhysicalElementComparator comparator)
{
  rmm::cuda_stream_view stream{cudf::get_default_stream()};

  auto const table_comparator =
    cudf::experimental::row::equality::two_table_comparator{lhs, rhs, stream};
  auto const equal_comparator =
    table_comparator.equal_to(cudf::nullate::NO{}, cudf::null_equality::EQUAL, comparator);
  auto const lhs_it = cudf::experimental::row::lhs_iterator(0);
  auto const rhs_it = cudf::experimental::row::rhs_iterator(0);

  auto output = cudf::make_numeric_column(
    cudf::data_type(cudf::type_id::BOOL8), lhs.num_rows(), cudf::mask_state::UNALLOCATED);

  thrust::transform(rmm::exec_policy(stream),
                    lhs_it,
                    lhs_it + lhs.num_rows(),
                    rhs_it,
                    output->mutable_view().data<bool>(),
                    equal_comparator);
  return output;
}

TYPED_TEST(TypedTableViewTest, TestLexicographicalComparatorTwoTables)
{
  using T = TypeParam;

  auto const col1         = cudf::test::fixed_width_column_wrapper<T>{{1, 2, 3, 4}};
  auto const col2         = cudf::test::fixed_width_column_wrapper<T>{{0, 1, 4, 3}};
  auto const column_order = std::vector{cudf::order::DESCENDING};
  auto const lhs          = cudf::table_view{{col1}};
  auto const rhs          = cudf::table_view{{col2}};

  auto const expected = cudf::test::fixed_width_column_wrapper<bool>{{1, 1, 0, 1}};
  auto const got      = two_table_comparison(
    lhs, rhs, column_order, cudf::experimental::row::lexicographic::physical_element_comparator{});
  CUDF_TEST_EXPECT_COLUMNS_EQUAL(expected, got->view());

  auto const sorting_got = two_table_comparison(
    lhs,
    rhs,
    column_order,
    cudf::experimental::row::lexicographic::sorting_physical_element_comparator{});
  CUDF_TEST_EXPECT_COLUMNS_EQUAL(expected, sorting_got->view());
}

TYPED_TEST(TypedTableViewTest, TestLexicographicalComparatorSameTable)
{
  using T = TypeParam;

  auto const col1         = cudf::test::fixed_width_column_wrapper<T>{{1, 2, 3, 4}};
  auto const column_order = std::vector{cudf::order::DESCENDING};
  auto const input_table  = cudf::table_view{{col1}};

  auto const expected = cudf::test::fixed_width_column_wrapper<bool>{{0, 0, 0, 0}};
  auto const got =
    self_comparison(input_table,
                    column_order,
                    cudf::experimental::row::lexicographic::physical_element_comparator{});
  CUDF_TEST_EXPECT_COLUMNS_EQUAL(expected, got->view());

  auto const sorting_got =
    self_comparison(input_table,
                    column_order,
                    cudf::experimental::row::lexicographic::sorting_physical_element_comparator{});
  CUDF_TEST_EXPECT_COLUMNS_EQUAL(expected, sorting_got->view());
}

template <typename T>
struct NaNTableViewTest : public cudf::test::BaseFixture {
};

TYPED_TEST_SUITE(NaNTableViewTest, cudf::test::FloatingPointTypes);

TYPED_TEST(NaNTableViewTest, TestLexicographicalComparatorTwoTableNaNCase)
{
  using T = TypeParam;

  auto const col1         = cudf::test::fixed_width_column_wrapper<T>{{T(NAN), T(NAN), T(1), T(1)}};
  auto const col2         = cudf::test::fixed_width_column_wrapper<T>{{T(NAN), T(1), T(NAN), T(1)}};
  auto const column_order = std::vector{cudf::order::DESCENDING};

  auto const lhs = cudf::table_view{{col1}};
  auto const rhs = cudf::table_view{{col2}};

  auto const expected = cudf::test::fixed_width_column_wrapper<bool>{{0, 0, 0, 0}};
  auto const got      = two_table_comparison(
    lhs, rhs, column_order, cudf::experimental::row::lexicographic::physical_element_comparator{});
  CUDF_TEST_EXPECT_COLUMNS_EQUAL(expected, got->view());

  auto const sorting_expected = cudf::test::fixed_width_column_wrapper<bool>{{0, 1, 0, 0}};
  auto const sorting_got      = two_table_comparison(
    lhs,
    rhs,
    column_order,
    cudf::experimental::row::lexicographic::sorting_physical_element_comparator{});
  CUDF_TEST_EXPECT_COLUMNS_EQUAL(sorting_expected, sorting_got->view());
}

TYPED_TEST(NaNTableViewTest, TestEqualityComparatorTwoTableNaNCase)
{
  using T = TypeParam;

  auto const col1         = cudf::test::fixed_width_column_wrapper<T>{{T(NAN), T(NAN), T(1), T(1)}};
  auto const col2         = cudf::test::fixed_width_column_wrapper<T>{{T(NAN), T(1), T(NAN), T(1)}};
  auto const column_order = std::vector{cudf::order::DESCENDING};

  auto const lhs = cudf::table_view{{col1}};
  auto const rhs = cudf::table_view{{col2}};

  auto const expected = cudf::test::fixed_width_column_wrapper<bool>{{0, 0, 0, 1}};
  auto const got      = two_table_equality(
    lhs, rhs, column_order, cudf::experimental::row::equality::physical_equality_comparator{});
  CUDF_TEST_EXPECT_COLUMNS_EQUAL(expected, got->view());

  auto const nan_equal_expected = cudf::test::fixed_width_column_wrapper<bool>{{1, 0, 0, 1}};
  auto const nan_equal_got =
    two_table_equality(lhs,
                       rhs,
                       column_order,
                       cudf::experimental::row::equality::nan_equal_physical_equality_comparator{});
  CUDF_TEST_EXPECT_COLUMNS_EQUAL(nan_equal_expected, nan_equal_got->view());
}
