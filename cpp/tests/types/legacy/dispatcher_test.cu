#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2018, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <cudf/cudf.h>
#include <tests/utilities/legacy/cudf_test_fixtures.h>
#include <cudf/utilities/legacy/type_dispatcher.hpp>

#include <thrust/device_vector.h>

#include <gtest/gtest.h>

#include <cstdint>

/**
 * @file dispatcher_test.cu
 * @brief Tests the type_dispatcher
 */

struct DispatcherTest : public GdfTest {
  /**---------------------------------------------------------------------------*
   * @brief Lists every gdf_dtype that the type_dispatcher supports.
   *
   * This vector *must* list every gdf_dtype supported by the type_dispatcher.
   *
   * If a new type gdf_dtype is added, but this list is not updated, then the
   * tests will fail.
   *
   *---------------------------------------------------------------------------**/
  std::vector<gdf_dtype> supported_dtypes{GDF_INT8,
                                          GDF_INT16,
                                          GDF_INT32,
                                          GDF_INT64,
                                          GDF_FLOAT32,
                                          GDF_FLOAT64,
                                          GDF_DATE32,
                                          GDF_DATE64,
                                          GDF_TIMESTAMP,
                                          GDF_CATEGORY,
                                          GDF_STRING_CATEGORY,
                                          GDF_BOOL8};

  // These types are not supported by the type_dispatcher
  std::vector<gdf_dtype> unsupported_dtypes{GDF_invalid, GDF_STRING};
};

using TestTypes = ::testing::Types<int8_t,
                                   int16_t,
                                   int32_t,
                                   int64_t,
                                   float,
                                   double,
                                   cudf::date32,
                                   cudf::date64,
                                   cudf::timestamp,
                                   cudf::category,
                                   cudf::nvstring_category,
                                   cudf::bool8>;

template <typename T>
struct TypedDispatcherTest : DispatcherTest {
};

TYPED_TEST_CASE(TypedDispatcherTest, TestTypes);

namespace {
template <typename ExpectedType>
struct type_tester {
  template <typename DispatchedType>
  bool operator()()
  {
    return std::is_same<ExpectedType, DispatchedType>::value;
  }
};
}  // namespace

// Ensure that the type_to_gdf_dtype trait maps to the correct gdf_dtype
TYPED_TEST(TypedDispatcherTest, TraitsTest)
{
  EXPECT_TRUE(cudf::type_dispatcher(cudf::gdf_dtype_of<TypeParam>(), type_tester<TypeParam>{}));
}

TEST_F(DispatcherTest, NumberOfTypesTest)
{
  // N_GDF_TYPES indicates how many enums there are in `gdf_dtype`,
  // therefore, if a gdf_dtype is added without updating this test, the test
  // will fail
  const size_t expected_num_supported_dtypes = N_GDF_TYPES - unsupported_dtypes.size();

  // Note: If this test fails, that means a type was added to gdf_dtype
  // without adding it to the `supported_dtypes` list in this test fixture
  ASSERT_EQ(expected_num_supported_dtypes, supported_dtypes.size())
    << "Number of supported types does not match what was expected.";
}

namespace {
struct test_functor {
  template <typename T>
  __host__ __device__ bool operator()(gdf_dtype type_id)
  {
    return (type_id == cudf::gdf_dtype_of<T>());
  }
};

__global__ void dispatch_test_kernel(gdf_dtype type, bool* d_result)
{
  if (0 == threadIdx.x + blockIdx.x * blockDim.x)
    *d_result = cudf::type_dispatcher(type, test_functor{}, type);
}
}  // namespace

// Every supported gdf_dtype should dispatch the correct type
TEST_F(DispatcherTest, HostDispatchFunctor)
{
  for (auto const& t : this->supported_dtypes) {
    bool result = cudf::type_dispatcher(t, test_functor{}, t);
    EXPECT_TRUE(result);
  }
}

TEST_F(DispatcherTest, DeviceDispatchFunctor)
{
  thrust::device_vector<bool> result(1);
  for (auto const& t : this->supported_dtypes) {
    dispatch_test_kernel<<<1, 1>>>(t, result.data().get());
    CUDA_TRY(hipDeviceSynchronize());
    EXPECT_EQ(true, result[0]);
  }
}

// Unsuported gdf_dtypes should throw std::runtime_error in host code
TEST_F(DispatcherTest, UnsuportedTypesTest)
{
  for (auto const& t : unsupported_dtypes) {
    EXPECT_THROW(cudf::type_dispatcher(t, test_functor{}, t), std::runtime_error);
  }
}

using DispatcherDeathTest = DispatcherTest;

// Unsuported gdf_dtypes in device code should set appropriate error code
// and invalidates device context
TEST_F(DispatcherDeathTest, DeviceDispatchFunctor)
{
  testing::FLAGS_gtest_death_test_style = "threadsafe";
  thrust::device_vector<bool> result(1);

  auto call_kernel = [&result](gdf_dtype t) {
    dispatch_test_kernel<<<1, 1>>>(t, result.data().get());
    auto error_code = hipDeviceSynchronize();

    // Kernel should fail with `hipErrorAssert` on an unsupported gdf_dtype
    // This error invalidates the current device context, so we need to kill
    // the current process. Running with EXPECT_DEATH spawns a new process for
    // each attempted kernel launch
    EXPECT_EQ(hipErrorAssert, error_code);
    exit(-1);
  };

  for (auto const& t : unsupported_dtypes) { EXPECT_DEATH(call_kernel(t), ""); }
}
