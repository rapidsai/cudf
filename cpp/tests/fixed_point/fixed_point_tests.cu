#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2020, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <cudf_test/base_fixture.hpp>
#include <cudf_test/column_wrapper.hpp>
#include <cudf_test/type_lists.hpp>

#include <cudf/column/column_factories.hpp>
#include <cudf/detail/utilities/cuda.cuh>
#include <cudf/fixed_point/fixed_point.hpp>
#include <cudf/null_mask.hpp>
#include <cudf/types.hpp>
#include <cudf/utilities/error.hpp>
#include <cudf/utilities/type_dispatcher.hpp>

#include <algorithm>
#include <limits>
#include <numeric>
#include <type_traits>
#include <vector>

using namespace numeric;

struct FixedPointTest : public cudf::test::BaseFixture {
};

template <typename T>
struct FixedPointTestBothReps : public cudf::test::BaseFixture {
};

using RepresentationTypes = ::testing::Types<int32_t, int64_t>;

TYPED_TEST_CASE(FixedPointTestBothReps, RepresentationTypes);

TYPED_TEST(FixedPointTestBothReps, SimpleDecimalXXConstruction)
{
  using decimalXX = fixed_point<TypeParam, Radix::BASE_10>;

  decimalXX num0{1.234567, scale_type{0}};
  decimalXX num1{1.234567, scale_type{-1}};
  decimalXX num2{1.234567, scale_type{-2}};
  decimalXX num3{1.234567, scale_type{-3}};
  decimalXX num4{1.234567, scale_type{-4}};
  decimalXX num5{1.234567, scale_type{-5}};
  decimalXX num6{1.234567, scale_type{-6}};

  EXPECT_EQ(1, static_cast<double>(num0));
  EXPECT_EQ(1.2, static_cast<double>(num1));
  EXPECT_EQ(1.23, static_cast<double>(num2));
  EXPECT_EQ(1.234, static_cast<double>(num3));
  EXPECT_EQ(1.2345, static_cast<double>(num4));
  EXPECT_EQ(1.23456, static_cast<double>(num5));
  EXPECT_EQ(1.234567, static_cast<double>(num6));
}

TYPED_TEST(FixedPointTestBothReps, SimpleNegativeDecimalXXConstruction)
{
  using decimalXX = fixed_point<TypeParam, Radix::BASE_10>;

  decimalXX num0{-1.234567, scale_type{0}};
  decimalXX num1{-1.234567, scale_type{-1}};
  decimalXX num2{-1.234567, scale_type{-2}};
  decimalXX num3{-1.234567, scale_type{-3}};
  decimalXX num4{-1.234567, scale_type{-4}};
  decimalXX num5{-1.234567, scale_type{-5}};
  decimalXX num6{-1.234567, scale_type{-6}};

  EXPECT_EQ(-1, static_cast<double>(num0));
  EXPECT_EQ(-1.2, static_cast<double>(num1));
  EXPECT_EQ(-1.23, static_cast<double>(num2));
  EXPECT_EQ(-1.234, static_cast<double>(num3));
  EXPECT_EQ(-1.2345, static_cast<double>(num4));
  EXPECT_EQ(-1.23456, static_cast<double>(num5));
  EXPECT_EQ(-1.234567, static_cast<double>(num6));
}

TYPED_TEST(FixedPointTestBothReps, PaddedDecimalXXConstruction)
{
  using decimalXX = fixed_point<TypeParam, Radix::BASE_10>;

  decimalXX a{1.1, scale_type{-1}};
  decimalXX b{1.01, scale_type{-2}};
  decimalXX c{1.001, scale_type{-3}};
  decimalXX d{1.0001, scale_type{-4}};
  decimalXX e{1.00001, scale_type{-5}};
  decimalXX f{1.000001, scale_type{-6}};

  decimalXX x{1.000123, scale_type{-8}};
  decimalXX y{0.000123, scale_type{-8}};

  EXPECT_EQ(1.1, static_cast<double>(a));
  EXPECT_EQ(1.01, static_cast<double>(b));
  EXPECT_EQ(1, static_cast<double>(c));  // intentional (inherited problem from floating point)
  EXPECT_EQ(1.0001, static_cast<double>(d));
  EXPECT_EQ(1.00001, static_cast<double>(e));
  EXPECT_EQ(1, static_cast<double>(f));  // intentional (inherited problem from floating point)

  EXPECT_TRUE(1.000123 - static_cast<double>(x) < std::numeric_limits<double>::epsilon());
  EXPECT_EQ(0.000123, static_cast<double>(y));
}

TYPED_TEST(FixedPointTestBothReps, SimpleBinaryFPConstruction)
{
  using binary_fp = fixed_point<TypeParam, Radix::BASE_2>;

  binary_fp num0{10, scale_type{0}};
  binary_fp num1{10, scale_type{1}};
  binary_fp num2{10, scale_type{2}};
  binary_fp num3{10, scale_type{3}};
  binary_fp num4{10, scale_type{4}};

  binary_fp num5{1.24, scale_type{0}};
  binary_fp num6{1.24, scale_type{-1}};
  binary_fp num7{1.32, scale_type{-2}};
  binary_fp num8{1.41, scale_type{-3}};
  binary_fp num9{1.45, scale_type{-4}};

  EXPECT_EQ(10, static_cast<double>(num0));
  EXPECT_EQ(10, static_cast<double>(num1));
  EXPECT_EQ(8, static_cast<double>(num2));
  EXPECT_EQ(8, static_cast<double>(num3));
  EXPECT_EQ(0, static_cast<double>(num4));

  EXPECT_EQ(1, static_cast<double>(num5));
  EXPECT_EQ(1, static_cast<double>(num6));
  EXPECT_EQ(1.25, static_cast<double>(num7));
  EXPECT_EQ(1.375, static_cast<double>(num8));
  EXPECT_EQ(1.4375, static_cast<double>(num9));
}

TYPED_TEST(FixedPointTestBothReps, MoreSimpleBinaryFPConstruction)
{
  using binary_fp = fixed_point<TypeParam, Radix::BASE_2>;

  binary_fp num0{1.25, scale_type{-2}};
  binary_fp num1{2.1, scale_type{-4}};

  EXPECT_EQ(1.25, static_cast<double>(num0));
  EXPECT_EQ(2.0625, static_cast<double>(num1));
}

TYPED_TEST(FixedPointTestBothReps, SimpleDecimalXXMath)
{
  using decimalXX = fixed_point<TypeParam, Radix::BASE_10>;

  decimalXX ONE{1, scale_type{-2}};
  decimalXX TWO{2, scale_type{-2}};
  decimalXX THREE{3, scale_type{-2}};
  decimalXX SIX{6, scale_type{-2}};

  EXPECT_TRUE(ONE + ONE == TWO);

  EXPECT_EQ(ONE + ONE, TWO);
  EXPECT_EQ(ONE * TWO, TWO);
  EXPECT_EQ(THREE * TWO, SIX);
  EXPECT_EQ(THREE - TWO, ONE);
  EXPECT_EQ(TWO / ONE, TWO);
  EXPECT_EQ(SIX / TWO, THREE);

  decimalXX a{1.23, scale_type{-2}};
  decimalXX b{0, scale_type{0}};

  EXPECT_EQ(a + b, a);
  EXPECT_EQ(a - b, a);
}

TYPED_TEST(FixedPointTestBothReps, ComparisonOperators)
{
  using decimalXX = fixed_point<TypeParam, Radix::BASE_10>;

  decimalXX ONE{1, scale_type{-1}};
  decimalXX TWO{2, scale_type{-2}};
  decimalXX THREE{3, scale_type{-3}};
  decimalXX SIX{6, scale_type{-4}};

  EXPECT_TRUE(ONE + ONE >= TWO);

  EXPECT_TRUE(ONE + ONE <= TWO);
  EXPECT_TRUE(ONE * TWO < THREE);
  EXPECT_TRUE(THREE * TWO > THREE);
  EXPECT_TRUE(THREE - TWO >= ONE);
  EXPECT_TRUE(TWO / ONE < THREE);
  EXPECT_TRUE(SIX / TWO >= ONE);
}

TYPED_TEST(FixedPointTestBothReps, DecimalXXTrickyDivision)
{
  using decimalXX = fixed_point<TypeParam, Radix::BASE_10>;

  decimalXX ONE_1{1, scale_type{1}};
  decimalXX SIX_0{6, scale_type{0}};
  decimalXX SIX_1{6, scale_type{1}};
  decimalXX TEN_0{10, scale_type{0}};
  decimalXX TEN_1{10, scale_type{1}};
  decimalXX SIXTY_1{60, scale_type{1}};

  EXPECT_EQ(static_cast<int32_t>(ONE_1), 0);
  EXPECT_EQ(static_cast<int32_t>(SIX_1), 0);
  EXPECT_EQ(static_cast<int32_t>(TEN_0), 10);
  EXPECT_EQ(static_cast<int32_t>(SIXTY_1), 60);

  EXPECT_EQ(SIXTY_1 / TEN_0, ONE_1);
  EXPECT_EQ(SIXTY_1 / TEN_1, SIX_0);

  decimalXX A{34.56, scale_type{-2}};
  decimalXX B{1.234, scale_type{-3}};
  decimalXX C{1, scale_type{-2}};

  EXPECT_EQ(static_cast<int32_t>(A / B), 20);
  EXPECT_EQ(static_cast<int32_t>((A * C) / B), 28);

  decimalXX n{28, scale_type{1}};
  EXPECT_EQ(static_cast<int32_t>(n), 20);
}

TYPED_TEST(FixedPointTestBothReps, DecimalXXRounding)
{
  using decimalXX = fixed_point<TypeParam, Radix::BASE_10>;

  decimalXX ZERO_0{0, scale_type{0}};
  decimalXX ZERO_1{4, scale_type{1}};
  decimalXX THREE_0{3, scale_type{0}};
  decimalXX FOUR_0{4, scale_type{0}};
  decimalXX FIVE_0{5, scale_type{0}};
  decimalXX TEN_0{10, scale_type{0}};
  decimalXX TEN_1{10, scale_type{1}};

  decimalXX FOURTEEN_0{14, scale_type{0}};
  decimalXX FIFTEEN_0{15, scale_type{0}};

  EXPECT_EQ(ZERO_0, ZERO_1);
  EXPECT_EQ(TEN_0, TEN_1);

  EXPECT_EQ(ZERO_1 + TEN_1, TEN_1);
  EXPECT_EQ(FOUR_0 + TEN_1, FOURTEEN_0);
  EXPECT_TRUE(ZERO_0 == ZERO_1);
  EXPECT_TRUE(FIVE_0 != TEN_1);
  EXPECT_TRUE(FIVE_0 + FIVE_0 + FIVE_0 == FIFTEEN_0);
  EXPECT_TRUE(FIVE_0 + FIVE_0 + FIVE_0 != TEN_1);
  EXPECT_TRUE(FIVE_0 * THREE_0 == FIFTEEN_0);
  EXPECT_TRUE(FIVE_0 * THREE_0 != TEN_1);
}

TYPED_TEST(FixedPointTestBothReps, ArithmeticWithDifferentScales)
{
  using decimalXX = fixed_point<TypeParam, Radix::BASE_10>;

  decimalXX a{1, scale_type{0}};
  decimalXX b{1.2, scale_type{-1}};
  decimalXX c{1.23, scale_type{-2}};
  decimalXX d{1.111, scale_type{-3}};

  decimalXX x{2.2, scale_type{-1}};
  decimalXX y{3.43, scale_type{-2}};
  decimalXX z{4.541, scale_type{-3}};

  decimalXX xx{0.2, scale_type{-1}};
  decimalXX yy{0.03, scale_type{-2}};
  decimalXX zz{0.119, scale_type{-3}};

  EXPECT_EQ(a + b, x);
  EXPECT_EQ(a + b + c, y);
  EXPECT_EQ(a + b + c + d, z);
  EXPECT_EQ(b - a, xx);
  EXPECT_EQ(c - b, yy);
  EXPECT_EQ(c - d, zz);
}

TYPED_TEST(FixedPointTestBothReps, RescaledTest)
{
  using decimalXX = fixed_point<TypeParam, Radix::BASE_10>;

  decimalXX num0{1, scale_type{0}};
  decimalXX num1{1.2, scale_type{-1}};
  decimalXX num2{1.23, scale_type{-2}};
  decimalXX num3{1.234, scale_type{-3}};
  decimalXX num4{1.2345, scale_type{-4}};
  decimalXX num5{1.23456, scale_type{-5}};
  decimalXX num6{1.234567, scale_type{-6}};

  EXPECT_EQ(num0, num6.rescaled(scale_type{0}));
  EXPECT_EQ(num1, num6.rescaled(scale_type{-1}));
  EXPECT_EQ(num2, num6.rescaled(scale_type{-2}));
  EXPECT_EQ(num3, num6.rescaled(scale_type{-3}));
  EXPECT_EQ(num4, num6.rescaled(scale_type{-4}));
  EXPECT_EQ(num5, num6.rescaled(scale_type{-5}));
}

TYPED_TEST(FixedPointTestBothReps, RescaledRounding)
{
  using decimalXX = fixed_point<TypeParam, Radix::BASE_10>;

  decimalXX num0{1500, scale_type{0}};
  decimalXX num1{1499, scale_type{0}};
  decimalXX num2{-1499, scale_type{0}};
  decimalXX num3{-1500, scale_type{0}};

  EXPECT_EQ(1000, static_cast<TypeParam>(num0.rescaled(scale_type{3})));
  EXPECT_EQ(1000, static_cast<TypeParam>(num1.rescaled(scale_type{3})));
  EXPECT_EQ(-1000, static_cast<TypeParam>(num2.rescaled(scale_type{3})));
  EXPECT_EQ(-1000, static_cast<TypeParam>(num3.rescaled(scale_type{3})));
}

TYPED_TEST(FixedPointTestBothReps, DecimalXXThrust)
{
  using decimalXX = fixed_point<TypeParam, Radix::BASE_10>;

  std::vector<decimalXX> vec1(1000);
  std::vector<int32_t> vec2(1000);

  std::iota(std::begin(vec1), std::end(vec1), decimalXX{0, scale_type{-2}});
  std::iota(std::begin(vec2), std::end(vec2), 0);

  auto const res1 =
    thrust::reduce(std::cbegin(vec1), std::cend(vec1), decimalXX{0, scale_type{-2}});

  auto const res2 = std::accumulate(std::cbegin(vec2), std::cend(vec2), 0);

  EXPECT_EQ(static_cast<int32_t>(res1), res2);

  std::vector<int32_t> vec3(vec1.size());

  thrust::transform(std::cbegin(vec1), std::cend(vec1), std::begin(vec3), [](auto const& e) {
    return static_cast<int32_t>(e);
  });

  EXPECT_EQ(vec2, vec3);
}

TYPED_TEST(FixedPointTestBothReps, BoolConversion)
{
  using decimalXX = fixed_point<TypeParam, Radix::BASE_10>;

  decimalXX truthy_value{1.234567, scale_type{0}};
  decimalXX falsy_value{0, scale_type{0}};

  // Test explicit conversions
  EXPECT_EQ(static_cast<bool>(truthy_value), true);
  EXPECT_EQ(static_cast<bool>(falsy_value), false);

  // These operators also *explicitly* convert to bool
  EXPECT_EQ(truthy_value && true, true);
  EXPECT_EQ(true && truthy_value, true);
  EXPECT_EQ(falsy_value || false, false);
  EXPECT_EQ(false || falsy_value, false);
  EXPECT_EQ(!truthy_value, false);
  EXPECT_EQ(!falsy_value, true);
}

TEST_F(FixedPointTest, OverflowDecimal32)
{
  // This flag is needed to avoid warnings with ASSERT_DEATH
  ::testing::FLAGS_gtest_death_test_style = "threadsafe";

  using decimal32 = fixed_point<int32_t, Radix::BASE_10>;

#if defined(__CUDACC_DEBUG__)

  decimal32 num0{2, scale_type{-9}};
  decimal32 num1{-2, scale_type{-9}};

  ASSERT_DEATH(num0 + num0, ".*");
  ASSERT_DEATH(num1 - num0, ".*");

  decimal32 min{std::numeric_limits<int32_t>::min(), scale_type{0}};
  decimal32 max{std::numeric_limits<int32_t>::max(), scale_type{0}};
  decimal32 NEG_ONE{-1, scale_type{0}};
  decimal32 ONE{1, scale_type{0}};
  decimal32 TWO{2, scale_type{0}};

  ASSERT_DEATH(min / NEG_ONE, ".*");
  ASSERT_DEATH(max * TWO, ".*");
  ASSERT_DEATH(min * TWO, ".*");
  ASSERT_DEATH(max + ONE, ".*");
  ASSERT_DEATH(max - NEG_ONE, ".*");
  ASSERT_DEATH(min - ONE, ".*");
  ASSERT_DEATH(max - NEG_ONE, ".*");

#endif
}

TEST_F(FixedPointTest, OverflowDecimal64)
{
  // This flag is needed to avoid warnings with ASSERT_DEATH
  ::testing::FLAGS_gtest_death_test_style = "threadsafe";

  using decimal64 = fixed_point<int64_t, Radix::BASE_10>;

#if defined(__CUDACC_DEBUG__)

  decimal64 num0{5, scale_type{-18}};
  decimal64 num1{-5, scale_type{-18}};

  ASSERT_DEATH(num0 + num0, ".*");
  ASSERT_DEATH(num1 - num0, ".*");

  decimal64 min{std::numeric_limits<int64_t>::min(), scale_type{0}};
  decimal64 max{std::numeric_limits<int64_t>::max(), scale_type{0}};
  decimal64 NEG_ONE{-1, scale_type{0}};
  decimal64 ONE{1, scale_type{0}};
  decimal64 TWO{2, scale_type{0}};

  ASSERT_DEATH(min / NEG_ONE, ".*");
  ASSERT_DEATH(max * TWO, ".*");
  ASSERT_DEATH(min * TWO, ".*");
  ASSERT_DEATH(max + ONE, ".*");
  ASSERT_DEATH(max - NEG_ONE, ".*");
  ASSERT_DEATH(min - ONE, ".*");
  ASSERT_DEATH(max - NEG_ONE, ".*");

#endif
}

template <typename ValueType, typename Binop>
void integer_vector_test(ValueType const initial_value,
                         int32_t const size,
                         int32_t const scale,
                         Binop binop)
{
  using decimal32 = fixed_point<int32_t, Radix::BASE_10>;

  std::vector<decimal32> vec1(size);
  std::vector<ValueType> vec2(size);

  std::iota(std::begin(vec1), std::end(vec1), decimal32{initial_value, scale_type{scale}});
  std::iota(std::begin(vec2), std::end(vec2), initial_value);

  auto const res1 =
    std::accumulate(std::cbegin(vec1), std::cend(vec1), decimal32{0, scale_type{scale}});

  auto const res2 = std::accumulate(std::cbegin(vec2), std::cend(vec2), static_cast<ValueType>(0));

  EXPECT_EQ(static_cast<int32_t>(res1), res2);

  std::vector<ValueType> vec3(vec1.size());

  std::transform(std::cbegin(vec1), std::cend(vec1), std::begin(vec3), [](auto const& e) {
    return static_cast<int32_t>(e);
  });

  EXPECT_EQ(vec2, vec3);
}

TEST_F(FixedPointTest, Decimal32IntVector)
{
  integer_vector_test(0, 10, -2, std::plus<>());
  integer_vector_test(0, 1000, -2, std::plus<>());

  integer_vector_test(1, 10, 0, std::multiplies<>());
  integer_vector_test(2, 20, 0, std::multiplies<>());
}

template <typename ValueType, typename Binop>
void float_vector_test(ValueType const initial_value,
                       int32_t const size,
                       int32_t const scale,
                       Binop binop)
{
  using decimal32 = fixed_point<int32_t, Radix::BASE_10>;

  std::vector<decimal32> vec1(size);
  std::vector<ValueType> vec2(size);

  std::iota(std::begin(vec1), std::end(vec1), decimal32{initial_value, scale_type{scale}});
  std::iota(std::begin(vec2), std::end(vec2), initial_value);

  auto equal = std::equal(
    std::cbegin(vec1), std::cend(vec1), std::cbegin(vec2), [](auto const& a, auto const& b) {
      return static_cast<double>(a) - b <= std::numeric_limits<ValueType>::epsilon();
    });

  EXPECT_TRUE(equal);
}

TEST_F(FixedPointTest, Decimal32FloatVector)
{
  float_vector_test(0.1, 1000, -2, std::plus<>());
  float_vector_test(0.15, 1000, -2, std::plus<>());

  float_vector_test(0.1, 10, -2, std::multiplies<>());
  float_vector_test(0.15, 20, -2, std::multiplies<>());
}

struct cast_to_int32_fn {
  using decimal32 = fixed_point<int32_t, Radix::BASE_10>;
  int32_t __host__ __device__ operator()(decimal32 fp) { return static_cast<int32_t>(fp); }
};

TEST_F(FixedPointTest, DecimalXXThrustOnDevice)
{
  using decimal32 = fixed_point<int32_t, Radix::BASE_10>;

  thrust::device_vector<decimal32> vec1(1000, decimal32{1, scale_type{-2}});

  auto const sum = thrust::reduce(
    rmm::exec_policy(0)->on(0), std::cbegin(vec1), std::cend(vec1), decimal32{0, scale_type{-2}});

  EXPECT_EQ(static_cast<int32_t>(sum), 1000);

  // TODO: Once nvbugs/1990211 is fixed (ExclusiveSum initial_value = 0 bug)
  //       change inclusive scan to run on device (avoid copying to host)
  thrust::host_vector<decimal32> vec1_host = vec1;

  thrust::inclusive_scan(std::cbegin(vec1_host), std::cend(vec1_host), std::begin(vec1_host));

  vec1 = vec1_host;

  std::vector<int32_t> vec2(1000);
  std::iota(std::begin(vec2), std::end(vec2), 1);

  auto const res1 = thrust::reduce(
    rmm::exec_policy(0)->on(0), std::cbegin(vec1), std::cend(vec1), decimal32{0, scale_type{-2}});

  auto const res2 = std::accumulate(std::cbegin(vec2), std::cend(vec2), 0);

  EXPECT_EQ(static_cast<int32_t>(res1), res2);

  thrust::device_vector<int32_t> vec3(1000);

  thrust::transform(rmm::exec_policy(0)->on(0),
                    std::cbegin(vec1),
                    std::cend(vec1),
                    std::begin(vec3),
                    cast_to_int32_fn{});

  thrust::host_vector<int32_t> vec3_host = vec3;

  EXPECT_EQ(vec2, vec3);
}

TYPED_TEST(FixedPointTestBothReps, FixedPointColumnWrapper)
{
  using namespace numeric;
  using decimalXX = fixed_point<TypeParam, Radix::BASE_10>;
  using RepType   = TypeParam;

  // fixed_point_column_wrapper
  auto const w = cudf::test::fixed_point_column_wrapper<RepType>{{1, 2, 3, 4}, scale_type{0}};

  // fixed_width_column_wrapper
  auto const ONE   = decimalXX{1, scale_type{0}};
  auto const TWO   = decimalXX{2, scale_type{0}};
  auto const THREE = decimalXX{3, scale_type{0}};
  auto const FOUR  = decimalXX{4, scale_type{0}};

  auto const vec = std::vector<decimalXX>{ONE, TWO, THREE, FOUR};
  auto const col = cudf::test::fixed_width_column_wrapper<decimalXX>(vec.begin(), vec.end());

  CUDF_TEST_EXPECT_COLUMNS_EQUAL(col, w);
}

TYPED_TEST(FixedPointTestBothReps, NoScaleOrWrongTypeID)
{
  auto const null_mask = cudf::create_null_mask(0, cudf::mask_state::ALL_NULL);

  EXPECT_THROW(cudf::make_fixed_point_column(cudf::data_type{cudf::type_id::INT32}, 0, null_mask),
               cudf::logic_error);
}

TYPED_TEST(FixedPointTestBothReps, SimpleFixedPointColumnWrapper)
{
  using decimalXX = fixed_point<TypeParam, Radix::BASE_10>;

  auto const a = cudf::test::fixed_point_column_wrapper<int32_t>{{11, 22, 33}, scale_type{-1}};
  auto const b = cudf::test::fixed_point_column_wrapper<int32_t>{{110, 220, 330}, scale_type{-2}};

  CUDF_TEST_EXPECT_COLUMNS_EQUAL(a, b);
}

CUDF_TEST_PROGRAM_MAIN()
