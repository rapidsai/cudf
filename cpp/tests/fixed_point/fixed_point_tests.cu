#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2020, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <algorithm>
#include <cudf/column/column_factories.hpp>
#include <cudf/fixed_point/fixed_point.hpp>
#include <cudf/utilities/type_dispatcher.hpp>
#include <limits>
#include <numeric>
#include <tests/utilities/base_fixture.hpp>
#include <tests/utilities/type_lists.hpp>
#include <type_traits>
#include <vector>

using namespace numeric;

struct FixedPointTest : public cudf::test::BaseFixture {
};

template <typename T>
struct FixedPointTestBothReps : public cudf::test::BaseFixture {
};

using RepresentationTypes = ::testing::Types<int32_t, int64_t>;

TYPED_TEST_CASE(FixedPointTestBothReps, RepresentationTypes);

TYPED_TEST(FixedPointTestBothReps, SimpleDecimalXXConstruction)
{
  using decimalXX = fixed_point<TypeParam, Radix::BASE_10>;

  decimalXX num0{1.234567, scale_type{0}};
  decimalXX num1{1.234567, scale_type{-1}};
  decimalXX num2{1.234567, scale_type{-2}};
  decimalXX num3{1.234567, scale_type{-3}};
  decimalXX num4{1.234567, scale_type{-4}};
  decimalXX num5{1.234567, scale_type{-5}};
  decimalXX num6{1.234567, scale_type{-6}};

  EXPECT_EQ(1, static_cast<double>(num0));
  EXPECT_EQ(1.2, static_cast<double>(num1));
  EXPECT_EQ(1.23, static_cast<double>(num2));
  EXPECT_EQ(1.235, static_cast<double>(num3));    // rounds up
  EXPECT_EQ(1.2346, static_cast<double>(num4));   // rounds up
  EXPECT_EQ(1.23457, static_cast<double>(num5));  // rounds up
  EXPECT_EQ(1.234567, static_cast<double>(num6));
}

TYPED_TEST(FixedPointTestBothReps, SimpleNegativeDecimalXXConstruction)
{
  using decimalXX = fixed_point<TypeParam, Radix::BASE_10>;

  decimalXX num0{-1.234567, scale_type{0}};
  decimalXX num1{-1.234567, scale_type{-1}};
  decimalXX num2{-1.234567, scale_type{-2}};
  decimalXX num3{-1.234567, scale_type{-3}};
  decimalXX num4{-1.234567, scale_type{-4}};
  decimalXX num5{-1.234567, scale_type{-5}};
  decimalXX num6{-1.234567, scale_type{-6}};

  EXPECT_EQ(-1, static_cast<double>(num0));
  EXPECT_EQ(-1.2, static_cast<double>(num1));
  EXPECT_EQ(-1.23, static_cast<double>(num2));
  EXPECT_EQ(-1.235, static_cast<double>(num3));    // rounds up
  EXPECT_EQ(-1.2346, static_cast<double>(num4));   // rounds up
  EXPECT_EQ(-1.23457, static_cast<double>(num5));  // rounds up
  EXPECT_EQ(-1.234567, static_cast<double>(num6));
}

TYPED_TEST(FixedPointTestBothReps, PaddedDecimalXXConstruction)
{
  using decimalXX = fixed_point<TypeParam, Radix::BASE_10>;

  decimalXX a{1.1, scale_type{-1}};
  decimalXX b{1.01, scale_type{-2}};
  decimalXX c{1.001, scale_type{-3}};
  decimalXX d{1.0001, scale_type{-4}};
  decimalXX e{1.00001, scale_type{-5}};
  decimalXX f{1.000001, scale_type{-6}};

  decimalXX x{1.000123, scale_type{-8}};
  decimalXX y{0.000123, scale_type{-8}};

  EXPECT_EQ(1.1, static_cast<double>(a));
  EXPECT_EQ(1.01, static_cast<double>(b));
  EXPECT_EQ(1.001, static_cast<double>(c));
  EXPECT_EQ(1.0001, static_cast<double>(d));
  EXPECT_EQ(1.00001, static_cast<double>(e));
  EXPECT_EQ(1.000001, static_cast<double>(f));

  EXPECT_TRUE(1.000123 - static_cast<double>(x) < std::numeric_limits<double>::epsilon());
  EXPECT_EQ(0.000123, static_cast<double>(y));
}

TYPED_TEST(FixedPointTestBothReps, SimpleBinaryFPConstruction)
{
  using binary_fp = fixed_point<TypeParam, Radix::BASE_2>;

  binary_fp num0{10, scale_type{0}};
  binary_fp num1{10, scale_type{1}};
  binary_fp num2{10, scale_type{2}};
  binary_fp num3{10, scale_type{3}};
  binary_fp num4{10, scale_type{4}};

  binary_fp num5{1.24, scale_type{0}};
  binary_fp num6{1.24, scale_type{-1}};
  binary_fp num7{1.32, scale_type{-2}};
  binary_fp num8{1.41, scale_type{-3}};
  binary_fp num9{1.45, scale_type{-4}};

  EXPECT_EQ(10, static_cast<double>(num0));
  EXPECT_EQ(10, static_cast<double>(num1));
  EXPECT_EQ(12, static_cast<double>(num2));
  EXPECT_EQ(8, static_cast<double>(num3));
  EXPECT_EQ(16, static_cast<double>(num4));

  EXPECT_EQ(1, static_cast<double>(num5));
  EXPECT_EQ(1, static_cast<double>(num6));
  EXPECT_EQ(1.25, static_cast<double>(num7));
  EXPECT_EQ(1.375, static_cast<double>(num8));
  EXPECT_EQ(1.4375, static_cast<double>(num9));
}

TYPED_TEST(FixedPointTestBothReps, MoreSimpleBinaryFPConstruction)
{
  using binary_fp = fixed_point<TypeParam, Radix::BASE_2>;

  binary_fp num0{1.25, scale_type{-2}};
  binary_fp num1{2.1, scale_type{-4}};

  EXPECT_EQ(1.25, static_cast<double>(num0));
  EXPECT_EQ(2.125, static_cast<double>(num1));
}

TYPED_TEST(FixedPointTestBothReps, SimpleDecimalXXMath)
{
  using decimalXX = fixed_point<TypeParam, Radix::BASE_10>;

  decimalXX ONE{1, scale_type{-2}};
  decimalXX TWO{2, scale_type{-2}};
  decimalXX THREE{3, scale_type{-2}};
  decimalXX SIX{6, scale_type{-2}};

  EXPECT_TRUE(ONE + ONE == TWO);

  EXPECT_EQ(ONE + ONE, TWO);
  EXPECT_EQ(ONE * TWO, TWO);
  EXPECT_EQ(THREE * TWO, SIX);
  EXPECT_EQ(THREE - TWO, ONE);
  EXPECT_EQ(TWO / ONE, TWO);
  EXPECT_EQ(SIX / TWO, THREE);
}

TYPED_TEST(FixedPointTestBothReps, DecimalXXTrickyDivision)
{
  using decimalXX = fixed_point<TypeParam, Radix::BASE_10>;

  decimalXX ONE_1{1, scale_type{1}};
  decimalXX SIX_0{6, scale_type{0}};
  decimalXX SIX_1{6, scale_type{1}};
  decimalXX TEN_0{10, scale_type{0}};
  decimalXX TEN_1{10, scale_type{1}};
  decimalXX SIXTY_1{60, scale_type{1}};

  EXPECT_EQ(static_cast<int32_t>(ONE_1), 0);   // round(1 / 10) = 0
  EXPECT_EQ(static_cast<int32_t>(SIX_1), 10);  // round(6 / 10) = 10
  EXPECT_EQ(static_cast<int32_t>(TEN_0), 10);
  EXPECT_EQ(static_cast<int32_t>(SIXTY_1), 60);

  EXPECT_EQ(SIXTY_1 / TEN_0, TEN_1);
  EXPECT_EQ(SIXTY_1 / TEN_1, SIX_0);

  decimalXX A{34.56, scale_type{-2}};
  decimalXX B{1.234, scale_type{-3}};
  decimalXX C{1, scale_type{-2}};

  EXPECT_EQ(static_cast<int32_t>(A / B), 30);
  EXPECT_EQ(static_cast<int32_t>((A * C) / B), 28);

  decimalXX n{28, scale_type{1}};
  EXPECT_EQ(static_cast<int32_t>(n), 30);
}

TYPED_TEST(FixedPointTestBothReps, DecimalXXRounding)
{
  using decimalXX = fixed_point<TypeParam, Radix::BASE_10>;

  decimalXX ZERO_FROM_FOUR_0{4, scale_type{0}};
  decimalXX ZERO_FROM_FOUR_1{4, scale_type{1}};
  decimalXX TEN_FROM_FIVE_0{5, scale_type{0}};
  decimalXX TEN_FROM_FIVE_1{5, scale_type{1}};

  EXPECT_EQ(ZERO_FROM_FOUR_1 + TEN_FROM_FIVE_1, TEN_FROM_FIVE_1);
  EXPECT_EQ(ZERO_FROM_FOUR_0 + TEN_FROM_FIVE_1, TEN_FROM_FIVE_1);
  EXPECT_TRUE(ZERO_FROM_FOUR_1 == ZERO_FROM_FOUR_1);
  EXPECT_TRUE(TEN_FROM_FIVE_0 == TEN_FROM_FIVE_1);
}

TYPED_TEST(FixedPointTestBothReps, DecimalXXThrust)
{
  using decimalXX = fixed_point<TypeParam, Radix::BASE_10>;

  std::vector<decimalXX> vec1(1000);
  std::vector<int32_t> vec2(1000);

  std::iota(std::begin(vec1), std::end(vec1), decimalXX{0, scale_type{-2}});
  std::iota(std::begin(vec2), std::end(vec2), 0);

  auto const res1 =
    thrust::reduce(std::cbegin(vec1), std::cend(vec1), decimalXX{0, scale_type{-2}});

  auto const res2 = std::accumulate(std::cbegin(vec2), std::cend(vec2), 0);

  EXPECT_EQ(static_cast<int32_t>(res1), res2);

  std::vector<int32_t> vec3(vec1.size());

  thrust::transform(std::cbegin(vec1), std::cend(vec1), std::begin(vec3), [](auto const& e) {
    return static_cast<int32_t>(e);
  });

  EXPECT_EQ(vec2, vec3);
}

TEST_F(FixedPointTest, OverflowDecimal32)
{
  using decimal32 = fixed_point<int32_t, Radix::BASE_10>;

#if defined(__CUDACC_DEBUG__)

  decimal32 num0{2, scale_type{-9}};
  decimal32 num1{-2, scale_type{-9}};

  ASSERT_NO_THROW(num0 + num0);
  ASSERT_NO_THROW(num1 - num0);

  decimal32 min{std::numeric_limits<int32_t>::min(), scale_type{0}};
  decimal32 max{std::numeric_limits<int32_t>::max(), scale_type{0}};
  decimal32 NEG_ONE{-1, scale_type{0}};
  decimal32 ONE{1, scale_type{0}};
  decimal32 TWO{2, scale_type{0}};

  ASSERT_NO_THROW(min / NEG_ONE);
  ASSERT_NO_THROW(max * TWO);
  ASSERT_NO_THROW(min * TWO);
  ASSERT_NO_THROW(max + ONE);
  ASSERT_NO_THROW(max - NEG_ONE);
  ASSERT_NO_THROW(min - ONE);
  ASSERT_NO_THROW(max - NEG_ONE);

#endif
}

TEST_F(FixedPointTest, OverflowDecimal64)
{
  using decimal64 = fixed_point<int64_t, Radix::BASE_10>;

#if defined(__CUDACC_DEBUG__)

  decimal64 num0{5, scale_type{-18}};
  decimal64 num1{-5, scale_type{-18}};

  ASSERT_NO_THROW(num0 + num0);
  ASSERT_NO_THROW(num1 - num0);

  decimal64 min{std::numeric_limits<int64_t>::min(), scale_type{0}};
  decimal64 max{std::numeric_limits<int64_t>::max(), scale_type{0}};
  decimal64 NEG_ONE{-1, scale_type{0}};
  decimal64 ONE{1, scale_type{0}};
  decimal64 TWO{2, scale_type{0}};

  ASSERT_NO_THROW(min / NEG_ONE);
  ASSERT_NO_THROW(max * TWO);
  ASSERT_NO_THROW(min * TWO);
  ASSERT_NO_THROW(max + ONE);
  ASSERT_NO_THROW(max - NEG_ONE);
  ASSERT_NO_THROW(min - ONE);
  ASSERT_NO_THROW(max - NEG_ONE);

#endif
}

template <typename ValueType, typename Binop>
void integer_vector_test(ValueType const initial_value,
                         int32_t const size,
                         int32_t const scale,
                         Binop binop)
{
  using decimal32 = fixed_point<int32_t, Radix::BASE_10>;

  std::vector<decimal32> vec1(size);
  std::vector<ValueType> vec2(size);

  std::iota(std::begin(vec1), std::end(vec1), decimal32{initial_value, scale_type{scale}});
  std::iota(std::begin(vec2), std::end(vec2), initial_value);

  auto const res1 =
    std::accumulate(std::cbegin(vec1), std::cend(vec1), decimal32{0, scale_type{scale}});

  auto const res2 = std::accumulate(std::cbegin(vec2), std::cend(vec2), static_cast<ValueType>(0));

  EXPECT_EQ(static_cast<int32_t>(res1), res2);

  std::vector<ValueType> vec3(vec1.size());

  std::transform(std::cbegin(vec1), std::cend(vec1), std::begin(vec3), [](auto const& e) {
    return static_cast<int32_t>(e);
  });

  EXPECT_EQ(vec2, vec3);
}

TEST_F(FixedPointTest, Decimal32IntVector)
{
  integer_vector_test(0, 10, -2, std::plus<>());
  integer_vector_test(0, 1000, -2, std::plus<>());

  integer_vector_test(1, 10, 0, std::multiplies<>());
  integer_vector_test(2, 20, 0, std::multiplies<>());
}

template <typename ValueType, typename Binop>
void float_vector_test(ValueType const initial_value,
                       int32_t const size,
                       int32_t const scale,
                       Binop binop)
{
  using decimal32 = fixed_point<int32_t, Radix::BASE_10>;

  std::vector<decimal32> vec1(size);
  std::vector<ValueType> vec2(size);

  std::iota(std::begin(vec1), std::end(vec1), decimal32{initial_value, scale_type{scale}});
  std::iota(std::begin(vec2), std::end(vec2), initial_value);

  auto equal = std::equal(
    std::cbegin(vec1), std::cend(vec1), std::cbegin(vec2), [](auto const& a, auto const& b) {
      return static_cast<double>(a) - b <= std::numeric_limits<ValueType>::epsilon();
    });

  EXPECT_TRUE(equal);
}

TEST_F(FixedPointTest, Decimal32FloatVector)
{
  float_vector_test(0.1, 1000, -2, std::plus<>());
  float_vector_test(0.15, 1000, -2, std::plus<>());

  float_vector_test(0.1, 10, -2, std::multiplies<>());
  float_vector_test(0.15, 20, -2, std::multiplies<>());
}

struct cast_to_int32_fn {
  using decimal32 = fixed_point<int32_t, Radix::BASE_10>;
  int32_t __host__ __device__ operator()(decimal32 fp) { return static_cast<int32_t>(fp); }
};

TEST_F(FixedPointTest, DecimalXXThrustOnDevice)
{
  using decimal32 = fixed_point<int32_t, Radix::BASE_10>;

  thrust::device_vector<decimal32> vec1(1000, decimal32{1, scale_type{-2}});

  auto const sum = thrust::reduce(
    rmm::exec_policy(0)->on(0), std::cbegin(vec1), std::cend(vec1), decimal32{0, scale_type{-2}});

  EXPECT_EQ(static_cast<int32_t>(sum), 1000);

  // TODO: Once nvbugs/1990211 is fixed (ExclusiveSum initial_value = 0 bug)
  //       change inclusive scan to run on device (avoid copying to host)
  thrust::host_vector<decimal32> vec1_host = vec1;

  thrust::inclusive_scan(std::cbegin(vec1_host), std::cend(vec1_host), std::begin(vec1_host));

  vec1 = vec1_host;

  std::vector<int32_t> vec2(1000);
  std::iota(std::begin(vec2), std::end(vec2), 1);

  auto const res1 = thrust::reduce(
    rmm::exec_policy(0)->on(0), std::cbegin(vec1), std::cend(vec1), decimal32{0, scale_type{-2}});

  auto const res2 = std::accumulate(std::cbegin(vec2), std::cend(vec2), 0);

  EXPECT_EQ(static_cast<int32_t>(res1), res2);

  thrust::device_vector<int32_t> vec3(1000);

  thrust::transform(rmm::exec_policy(0)->on(0),
                    std::cbegin(vec1),
                    std::cend(vec1),
                    std::begin(vec3),
                    cast_to_int32_fn{});

  thrust::host_vector<int32_t> vec3_host = vec3;

  EXPECT_EQ(vec2, vec3);
}

CUDF_TEST_PROGRAM_MAIN()
