/*
 * Copyright (c) 2019, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <tests/utilities/legacy/cudf_test_fixtures.h>
#include <cudf/legacy/groupby.hpp>
#include <cudf/legacy/table.hpp>
#include <cudf/utilities/legacy/type_dispatcher.hpp>
#include <tests/utilities/legacy/column_wrapper.cuh>
#include <tests/utilities/legacy/compare_column_wrappers.cuh>
#include "../../common/legacy/type_info.hpp"
#include "single_column_groupby_test.cuh"

#include <gmock/gmock.h>
#include <gtest/gtest.h>

#include <random>

static constexpr cudf::groupby::operators op{cudf::groupby::operators::COUNT};

template <typename T>
struct SingleColumnCount : public GdfTest {
  using KeyType = T;

  // For COUNT, the value type doesn't matter
  using ValueType = int;
};

template <typename T>
using column_wrapper = cudf::test::column_wrapper<T>;

// TODO: tests for cudf::bool8
using TestingTypes = ::testing::Types<int32_t,
                                      int8_t,
                                      int16_t,
                                      int32_t,
                                      int64_t,
                                      float,
                                      double,
                                      cudf::date32,
                                      cudf::date64,
                                      cudf::category>;

TYPED_TEST_CASE(SingleColumnCount, TestingTypes);

TYPED_TEST(SingleColumnCount, OneGroupNoNulls)
{
  constexpr int size{10};
  using Key         = typename SingleColumnCount<TypeParam>::KeyType;
  using Value       = typename SingleColumnCount<TypeParam>::ValueType;
  using ResultValue = cudf::test::expected_result_t<Value, op>;
  TypeParam key{42};
  cudf::test::single_column_groupby_test<op>(
    column_wrapper<Key>(size, [key](auto index) { return key; }),
    column_wrapper<Value>(size, [](auto index) { return Value(index); }),
    column_wrapper<Key>{TypeParam(42)},
    column_wrapper<ResultValue>{size});
}

TYPED_TEST(SingleColumnCount, OneGroupAllNullKeys)
{
  constexpr int size{10};
  using Key         = typename SingleColumnCount<TypeParam>::KeyType;
  using Value       = typename SingleColumnCount<TypeParam>::ValueType;
  using ResultValue = cudf::test::expected_result_t<Value, op>;
  TypeParam key{42};

  // If all keys are null, then there should be no output
  cudf::test::single_column_groupby_test<op>(
    column_wrapper<Key>(
      size, [key](auto index) { return key; }, [](auto index) { return false; }),
    column_wrapper<Value>(size, [](auto index) { return Value(index); }),
    column_wrapper<Key>{},
    column_wrapper<ResultValue>{});
}

TYPED_TEST(SingleColumnCount, OneGroupAllNullValues)
{
  constexpr int size{10};
  using Key         = typename SingleColumnCount<TypeParam>::KeyType;
  using Value       = typename SingleColumnCount<TypeParam>::ValueType;
  using ResultValue = cudf::test::expected_result_t<Value, op>;
  TypeParam key{42};
  // If all values are null, then the output count should be a non-null zero
  cudf::test::single_column_groupby_test<op>(
    column_wrapper<Key>(size, [key](auto index) { return key; }),
    column_wrapper<Value>(
      size, [](auto index) { return Value(index); }, [](auto index) { return false; }),
    column_wrapper<Key>({key}),
    column_wrapper<ResultValue>({0}, [](auto index) { return true; }));
}

TYPED_TEST(SingleColumnCount, OneGroupEvenNullKeys)
{
  constexpr int size{10};
  using Key         = typename SingleColumnCount<TypeParam>::KeyType;
  using Value       = typename SingleColumnCount<TypeParam>::ValueType;
  using ResultValue = cudf::test::expected_result_t<Value, op>;
  TypeParam key{42};

  EXPECT_EQ(size % 2, 0) << "Size must be multiple of 2 for this test.";
  // Odd index keys are null, means COUNT should be size/2
  // Output keys should be nullable
  cudf::test::single_column_groupby_test<op>(
    column_wrapper<Key>(
      size, [key](auto index) { return key; }, [](auto index) { return index % 2; }),
    column_wrapper<Value>(size, [](auto index) { return Value(index); }),
    column_wrapper<Key>({key}, [](auto index) { return true; }),
    column_wrapper<ResultValue>({size / 2}));
}

TYPED_TEST(SingleColumnCount, OneGroupEvenNullValues)
{
  constexpr int size{10};
  using Key         = typename SingleColumnCount<TypeParam>::KeyType;
  using Value       = typename SingleColumnCount<TypeParam>::ValueType;
  using ResultValue = cudf::test::expected_result_t<Value, op>;
  TypeParam key{42};

  EXPECT_EQ(size % 2, 0) << "Size must be multiple of 2 for this test.";
  // Odd index values are null, means COUNT should be size/2
  // Output values should be nullable
  cudf::test::single_column_groupby_test<op>(
    column_wrapper<Key>(size, [key](auto index) { return key; }),
    column_wrapper<Value>(
      size, [](auto index) { return Value(index); }, [](auto index) { return index % 2; }),
    column_wrapper<Key>({key}),
    column_wrapper<ResultValue>({size / 2}, [](auto index) { return true; }));
}

TYPED_TEST(SingleColumnCount, FourGroupsNoNulls)
{
  using Key         = typename SingleColumnCount<TypeParam>::KeyType;
  using Value       = typename SingleColumnCount<TypeParam>::ValueType;
  using ResultValue = cudf::test::expected_result_t<Value, op>;
  using T           = Key;
  using R           = ResultValue;

  // Each value needs to be casted to avoid a narrowing conversion warning for
  // the wrapper types
  cudf::test::single_column_groupby_test<op>(
    column_wrapper<Key>{T(1), T(1), T(2), T(2), T(3), T(3), T(4), T(4)},
    column_wrapper<Value>(8, [](auto index) { return Value(index); }),
    column_wrapper<Key>{T(1), T(2), T(3), T(4)},
    column_wrapper<ResultValue>{R(2), R(2), R(2), R(2)});
}

TYPED_TEST(SingleColumnCount, FourGroupsEvenNullKeys)
{
  using Key         = typename SingleColumnCount<TypeParam>::KeyType;
  using Value       = typename SingleColumnCount<TypeParam>::ValueType;
  using ResultValue = cudf::test::expected_result_t<Value, op>;
  using T           = Key;
  using R           = ResultValue;

  // Odd index keys are null, COUNT should be the count of each key / 2
  // Output keys should be nullable
  cudf::test::single_column_groupby_test<op>(
    column_wrapper<Key>({T(1), T(1), T(2), T(2), T(3), T(3), T(4), T(4)},
                        [](auto index) { return index % 2; }),
    column_wrapper<Value>(8, [](auto index) { return Value(index); }),
    column_wrapper<Key>({T(1), T(2), T(3), T(4)}, [](auto index) { return true; }),
    column_wrapper<ResultValue>{R(1), R(1), R(1), R(1)});
}

TYPED_TEST(SingleColumnCount, FourGroupsEvenNullValues)
{
  using Key         = typename SingleColumnCount<TypeParam>::KeyType;
  using Value       = typename SingleColumnCount<TypeParam>::ValueType;
  using ResultValue = cudf::test::expected_result_t<Value, op>;
  using T           = Key;
  using R           = ResultValue;

  // Odd index values are null, COUNT should be the count of each key / 2
  // Output values should be nullable
  cudf::test::single_column_groupby_test<op>(
    column_wrapper<Key>{T(1), T(1), T(2), T(2), T(3), T(3), T(4), T(4)},
    column_wrapper<Value>(
      8, [](auto index) { return Value(index); }, [](auto index) { return index % 2; }),
    column_wrapper<Key>({T(1), T(2), T(3), T(4)}),
    column_wrapper<ResultValue>({R(1), R(1), R(1), R(1)}, [](auto index) { return true; }));
}

TYPED_TEST(SingleColumnCount, FourGroupsEvenNullValuesKeys)
{
  using Key         = typename SingleColumnCount<TypeParam>::KeyType;
  using Value       = typename SingleColumnCount<TypeParam>::ValueType;
  using ResultValue = cudf::test::expected_result_t<Value, op>;
  using T           = Key;
  using R           = ResultValue;

  // Odd index keys and values are null,
  //  COUNT should be the count of each key / 2 Output keys and values should be
  //  nullable
  cudf::test::single_column_groupby_test<op>(
    column_wrapper<Key>({T(1), T(1), T(2), T(2), T(3), T(3), T(4), T(4)},
                        [](auto index) { return index % 2; }),
    column_wrapper<Value>(
      8, [](auto index) { return Value(index); }, [](auto index) { return index % 2; }),
    column_wrapper<Key>({T(1), T(2), T(3), T(4)}, [](auto index) { return true; }),
    column_wrapper<ResultValue>({R(1), R(1), R(1), R(1)}, [](auto index) { return true; }));
}

TYPED_TEST(SingleColumnCount, FourGroupsEvenNullValuesOddNullKeys)
{
  using Key         = typename SingleColumnCount<TypeParam>::KeyType;
  using Value       = typename SingleColumnCount<TypeParam>::ValueType;
  using ResultValue = cudf::test::expected_result_t<Value, op>;
  using T           = Key;
  using R           = ResultValue;

  // Even keys are null & Odd values are null
  // Count for each key should thefore be 0
  cudf::test::single_column_groupby_test<op>(
    column_wrapper<Key>({T(1), T(1), T(2), T(2), T(3), T(3), T(4), T(4)},
                        [](auto index) { return not(index % 2); }),
    column_wrapper<Value>(
      8, [](auto index) { return Value(index); }, [](auto index) { return index % 2; }),
    column_wrapper<Key>({T(1), T(2), T(3), T(4)}, [](auto index) { return true; }),
    column_wrapper<ResultValue>({R(0), R(0), R(0), R(0)}, [](auto index) { return true; }));
}
