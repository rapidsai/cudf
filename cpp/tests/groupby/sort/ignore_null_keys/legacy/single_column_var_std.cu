#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <tests/utilities/legacy/cudf_test_fixtures.h>
#include <cudf/legacy/groupby.hpp>
#include <cudf/legacy/table.hpp>
#include <cudf/utilities/legacy/type_dispatcher.hpp>
#include <tests/groupby/common/legacy/type_info.hpp>
#include <tests/groupby/sort/legacy/single_column_groupby_test.cuh>
#include <tests/utilities/legacy/column_wrapper.cuh>
#include <tests/utilities/legacy/compare_column_wrappers.cuh>

#include <gmock/gmock.h>
#include <gtest/gtest.h>

#include <random>

static constexpr cudf::groupby::operators var_op{cudf::groupby::operators::VARIANCE};

static constexpr cudf::groupby::operators std_op{cudf::groupby::operators::STD};

template <typename KV>
struct SingleColumnVarStd : public GdfTest {
  using KeyType   = typename KV::Key;
  using ValueType = typename KV::Value;
};

template <typename K>
struct key_accessor {
  std::vector<int> const& keys;
  K operator()(cudf::size_type i) { return K(keys[i]); }
};

template <typename T>
using column_wrapper = cudf::test::column_wrapper<T>;

template <typename K, typename V>
struct KV {
  using Key   = K;
  using Value = V;
};

using TestingTypes = ::testing::Types<KV<int8_t, int8_t>,
                                      KV<int32_t, int32_t>,
                                      KV<int64_t, int64_t>,
                                      KV<int32_t, float>,
                                      KV<int32_t, double>,
                                      KV<cudf::category, int32_t>,
                                      KV<cudf::date32, int8_t>,
                                      KV<cudf::date64, double>>;

// TODO: tests for cudf::bool8

using std_args = cudf::groupby::sort::std_args;

TYPED_TEST_CASE(SingleColumnVarStd, TestingTypes);

TYPED_TEST(SingleColumnVarStd, TestVarStdPreSorted)
{
  using K = typename SingleColumnVarStd<TypeParam>::KeyType;
  using V = typename SingleColumnVarStd<TypeParam>::ValueType;
  using R = cudf::test::expected_result_t<V, var_op>;
  using T = int;

  std::vector<T> in_keys{1, 1, 1, 2, 2, 2, 2, 2};
  std::vector<V> in_vals{0, 1, 2, 3, 4, 5, 6, 7};

  //{1, 1, 1, 2, 2, 2, 2, 2}
  std::vector<T> out_keys{
    1,
    2,
  };
  //{0, 1, 2, 3, 4, 5, 6, 7}
  std::vector<R> out_vals{1, 2.5};

  int ddof = 1;
  cudf::test::single_column_groupby_test<var_op>(
    {var_op, std::make_unique<std_args>(ddof)},
    column_wrapper<K>(in_keys.size(), key_accessor<K>{in_keys}),
    column_wrapper<V>(in_vals),
    column_wrapper<K>(out_keys.size(), key_accessor<K>{out_keys}),
    column_wrapper<R>(out_vals, [](auto) { return true; }));

  cudf::test::single_column_groupby_test<std_op>(
    {std_op, std::make_unique<std_args>(ddof)},
    column_wrapper<K>(in_keys.size(), key_accessor<K>{in_keys}),
    column_wrapper<V>(in_vals),
    column_wrapper<K>(out_keys.size(), key_accessor<K>{out_keys}),
    column_wrapper<R>(
      out_vals.size(), [&](auto i) { return sqrt(out_vals[i]); }, [](auto) { return true; }));
}

TYPED_TEST(SingleColumnVarStd, TestVarStd)
{
  using K = typename SingleColumnVarStd<TypeParam>::KeyType;
  using V = typename SingleColumnVarStd<TypeParam>::ValueType;
  using R = cudf::test::expected_result_t<V, var_op>;
  using T = int;

  std::vector<T> in_keys{3, 2, 1, 1, 2, 3, 3, 2, 1};
  std::vector<V> in_vals{1, 2, 3, 4, 4, 3, 2, 1, 0};

  //{1, 1, 1, 2, 2, 2, 3, 3, 3}
  std::vector<T> out_keys{1, 2, 3};
  //{0, 3, 4, 1, 2, 4, 1, 2, 3}
  std::vector<R> out_vals{13. / 3, 7. / 3, 1};

  int ddof = 1;
  cudf::test::single_column_groupby_test<var_op>(
    {var_op, std::make_unique<std_args>(ddof)},
    column_wrapper<K>(in_keys.size(), key_accessor<K>{in_keys}),
    column_wrapper<V>(in_vals),
    column_wrapper<K>(out_keys.size(), key_accessor<K>{out_keys}),
    column_wrapper<R>(out_vals, [](auto) { return true; }));

  cudf::test::single_column_groupby_test<std_op>(
    {std_op, std::make_unique<std_args>(ddof)},
    column_wrapper<K>(in_keys.size(), key_accessor<K>{in_keys}),
    column_wrapper<V>(in_vals),
    column_wrapper<K>(out_keys.size(), key_accessor<K>{out_keys}),
    column_wrapper<R>(
      out_vals.size(), [&](auto i) { return sqrt(out_vals[i]); }, [](auto) { return true; }));
}

TYPED_TEST(SingleColumnVarStd, TestVarStdDifferentSizeGroups)
{
  using K = typename SingleColumnVarStd<TypeParam>::KeyType;
  using V = typename SingleColumnVarStd<TypeParam>::ValueType;
  using R = cudf::test::expected_result_t<V, var_op>;
  using T = int;

  std::vector<T> in_keys{1, 2, 3, 3, 2, 1, 0, 3, 0, 1, 0, 2, 3, 0, 3, 3, 2, 1, 0};
  std::vector<V> in_vals{0, 1, 2, 3, 4, 5, 6, 7, 8, 9, 8, 7, 6, 5, 4, 3, 2, 1, 0};

  //{0, 0, 0, 0, 0, 1, 1, 1, 1, 2, 2, 2, 2, 3, 3, 3, 3, 3, 3}
  std::vector<T> out_keys{0, 1, 2, 3};
  //{0, 5, 6, 8, 8, 0, 1, 5, 9, 1, 2, 4, 7, 2, 3, 3, 4, 6, 7}
  std::vector<R> out_vals{10.8, 203. / 12, 7, 113. / 30};

  int ddof = 1;
  cudf::test::single_column_groupby_test<var_op>(
    {var_op, std::make_unique<std_args>(ddof)},
    column_wrapper<K>(in_keys.size(), key_accessor<K>{in_keys}),
    column_wrapper<V>(in_vals),
    column_wrapper<K>(out_keys.size(), key_accessor<K>{out_keys}),
    column_wrapper<R>(out_vals, [](auto) { return true; }));

  cudf::test::single_column_groupby_test<std_op>(
    {std_op, std::make_unique<std_args>(ddof)},
    column_wrapper<K>(in_keys.size(), key_accessor<K>{in_keys}),
    column_wrapper<V>(in_vals),
    column_wrapper<K>(out_keys.size(), key_accessor<K>{out_keys}),
    column_wrapper<R>(
      out_vals.size(), [&](auto i) { return sqrt(out_vals[i]); }, [](auto) { return true; }));
}

TYPED_TEST(SingleColumnVarStd, TestVarStdNullable)
{
  using K = typename SingleColumnVarStd<TypeParam>::KeyType;
  using V = typename SingleColumnVarStd<TypeParam>::ValueType;
  using R = cudf::test::expected_result_t<V, var_op>;
  using T = int;

  std::vector<T> in_keys{1, 1, 1, 1, 1};
  std::vector<bool> key_valid{1, 0, 1, 1, 1};
  std::vector<V> in_vals{0, 1, 2, 3, 4};
  std::vector<bool> vals_valid{1, 1, 0, 1, 1};

  //{1, -, 1, 1, 1}
  std::vector<T> out_keys{
    1,
  };
  //{0, 1, -, 3, 4}
  std::vector<R> out_vals{
    13. / 3,
  };
  std::vector<bool> out_valids{
    1,
  };

  int ddof = 1;
  cudf::test::single_column_groupby_test<var_op>(
    {var_op, std::make_unique<std_args>(ddof)},
    column_wrapper<K>(
      in_keys.size(), key_accessor<K>{in_keys}, [&](auto index) { return key_valid[index]; }),
    column_wrapper<V>(in_vals, [&](auto index) { return vals_valid[index]; }),
    column_wrapper<K>(out_keys.size(), key_accessor<K>{out_keys}, [](auto index) { return true; }),
    column_wrapper<R>(out_vals, [&](auto index) { return out_valids[index]; }));

  cudf::test::single_column_groupby_test<std_op>(
    {std_op, std::make_unique<std_args>(ddof)},
    column_wrapper<K>(
      in_keys.size(), key_accessor<K>{in_keys}, [&](auto index) { return key_valid[index]; }),
    column_wrapper<V>(in_vals, [&](auto index) { return vals_valid[index]; }),
    column_wrapper<K>(out_keys.size(), key_accessor<K>{out_keys}, [](auto index) { return true; }),
    column_wrapper<R>(
      out_vals.size(),
      [&](auto index) { return sqrt(out_vals[index]); },
      [&](auto index) { return out_valids[index]; }));
}

TYPED_TEST(SingleColumnVarStd, TestVarStdNullableZeroGroupSize)
{
  using K = typename SingleColumnVarStd<TypeParam>::KeyType;
  using V = typename SingleColumnVarStd<TypeParam>::ValueType;
  using R = cudf::test::expected_result_t<V, var_op>;
  using T = int;

  std::vector<T> in_keys{1, 1, 1, 1, 1, 2, 2};
  std::vector<bool> key_valid{1, 0, 1, 1, 1, 1, 1};
  std::vector<V> in_vals{0, 1, 2, 3, 4, 5, 6};
  std::vector<bool> vals_valid{1, 1, 0, 1, 1, 0, 0};

  //{1, -, 1, 1, 1, 2, 2}
  std::vector<T> out_keys{1, 2};
  //{0, 1, -, 3, 4, -, 0}
  std::vector<R> out_vals{
    13. / 3,
    0,
  };
  std::vector<bool> out_valids{
    1,
    0,
  };

  int ddof = 1;
  cudf::test::single_column_groupby_test<var_op>(
    {var_op, std::make_unique<std_args>(ddof)},
    column_wrapper<K>(
      in_keys.size(), key_accessor<K>{in_keys}, [&](auto index) { return key_valid[index]; }),
    column_wrapper<V>(in_vals, [&](auto index) { return vals_valid[index]; }),
    column_wrapper<K>(out_keys.size(), key_accessor<K>{out_keys}, [](auto index) { return true; }),
    column_wrapper<R>(out_vals, [&](auto index) { return out_valids[index]; }));

  cudf::test::single_column_groupby_test<std_op>(
    {std_op, std::make_unique<std_args>(ddof)},
    column_wrapper<K>(
      in_keys.size(), key_accessor<K>{in_keys}, [&](auto index) { return key_valid[index]; }),
    column_wrapper<V>(in_vals, [&](auto index) { return vals_valid[index]; }),
    column_wrapper<K>(out_keys.size(), key_accessor<K>{out_keys}, [](auto index) { return true; }),
    column_wrapper<R>(
      out_vals.size(),
      [&](auto index) { return sqrt(out_vals[index]); },
      [&](auto index) { return out_valids[index]; }));
}

TYPED_TEST(SingleColumnVarStd, TestVarStdNullableZeroDDoFDivisor)
{
  using K = typename SingleColumnVarStd<TypeParam>::KeyType;
  using V = typename SingleColumnVarStd<TypeParam>::ValueType;
  using R = cudf::test::expected_result_t<V, var_op>;
  using T = int;

  std::vector<T> in_keys{1, 1, 1, 1, 1, 3};
  std::vector<bool> key_valid{1, 0, 1, 1, 1, 1};
  std::vector<V> in_vals{0, 1, 2, 3, 4, 7};
  std::vector<bool> vals_valid{1, 1, 0, 1, 1, 1};

  //{1, -, 1, 1, 1, 3}
  std::vector<T> out_keys{1, 3};
  //{0, 1, -, 3, 4, 7}
  std::vector<R> out_vals{13. / 3, 0};
  std::vector<bool> out_valids{1, 0};

  int ddof = 1;
  cudf::test::single_column_groupby_test<var_op>(
    {var_op, std::make_unique<std_args>(ddof)},
    column_wrapper<K>(
      in_keys.size(), key_accessor<K>{in_keys}, [&](auto index) { return key_valid[index]; }),
    column_wrapper<V>(in_vals, [&](auto index) { return vals_valid[index]; }),
    column_wrapper<K>(out_keys.size(), key_accessor<K>{out_keys}, [](auto index) { return true; }),
    column_wrapper<R>(out_vals, [&](auto index) { return out_valids[index]; }));

  cudf::test::single_column_groupby_test<std_op>(
    {std_op, std::make_unique<std_args>(ddof)},
    column_wrapper<K>(
      in_keys.size(), key_accessor<K>{in_keys}, [&](auto index) { return key_valid[index]; }),
    column_wrapper<V>(in_vals, [&](auto index) { return vals_valid[index]; }),
    column_wrapper<K>(out_keys.size(), key_accessor<K>{out_keys}, [](auto index) { return true; }),
    column_wrapper<R>(
      out_vals.size(),
      [&](auto index) { return sqrt(out_vals[index]); },
      [&](auto index) { return out_valids[index]; }));
}
