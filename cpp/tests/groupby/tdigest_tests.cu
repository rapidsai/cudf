#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2021-2022, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <cudf/detail/aggregation/aggregation.hpp>
#include <cudf/detail/tdigest/tdigest.hpp>
#include <cudf/lists/lists_column_view.hpp>
#include <cudf/tdigest/tdigest_column_view.hpp>
#include <cudf/utilities/default_stream.hpp>

#include <cudf_test/base_fixture.hpp>
#include <cudf_test/column_wrapper.hpp>
#include <cudf_test/iterator_utilities.hpp>
#include <cudf_test/tdigest_utilities.cuh>
#include <cudf_test/type_lists.hpp>

#include <thrust/copy.h>
#include <thrust/fill.h>
#include <thrust/iterator/counting_iterator.h>

namespace cudf {
namespace test {

using namespace cudf;

/**
 * @brief Functor to generate a tdigest by key.
 *
 */
struct tdigest_gen_grouped {
  template <
    typename T,
    typename std::enable_if_t<cudf::is_numeric<T>() || cudf::is_fixed_point<T>()>* = nullptr>
  std::unique_ptr<column> operator()(column_view const& keys, column_view const& values, int delta)
  {
    cudf::table_view t({keys});
    cudf::groupby::groupby gb(t);
    std::vector<cudf::groupby::aggregation_request> requests;
    std::vector<std::unique_ptr<cudf::groupby_aggregation>> aggregations;
    aggregations.push_back(cudf::make_tdigest_aggregation<cudf::groupby_aggregation>(delta));
    requests.push_back({values, std::move(aggregations)});
    auto result = gb.aggregate(requests);
    return std::move(result.second[0].results[0]);
  }

  template <
    typename T,
    typename std::enable_if_t<!cudf::is_numeric<T>() && !cudf::is_fixed_point<T>()>* = nullptr>
  std::unique_ptr<column> operator()(column_view const& keys, column_view const& values, int delta)
  {
    CUDF_FAIL("Invalid tdigest test type");
  }
};

/**
 * @brief Functor for generating a tdigest using groupby with a constant key.
 *
 */
struct tdigest_groupby_simple_op {
  std::unique_ptr<column> operator()(column_view const& values, int delta) const
  {
    // make a simple set of matching keys.
    auto keys = cudf::make_fixed_width_column(
      data_type{type_id::INT32}, values.size(), mask_state::UNALLOCATED);
    thrust::fill(rmm::exec_policy(cudf::get_default_stream()),
                 keys->mutable_view().template begin<int>(),
                 keys->mutable_view().template end<int>(),
                 0);

    cudf::table_view t({*keys});
    cudf::groupby::groupby gb(t);
    std::vector<cudf::groupby::aggregation_request> requests;
    std::vector<std::unique_ptr<cudf::groupby_aggregation>> aggregations;
    aggregations.push_back(cudf::make_tdigest_aggregation<cudf::groupby_aggregation>(delta));
    requests.push_back({values, std::move(aggregations)});
    auto result = gb.aggregate(requests);
    return std::move(result.second[0].results[0]);
  }
};

/**
 * @brief Functor for merging tdigests using groupby with a constant key.
 *
 */
struct tdigest_groupby_simple_merge_op {
  std::unique_ptr<column> operator()(column_view const& merge_values, int merge_delta) const
  {
    // make a simple set of matching keys.
    auto merge_keys = cudf::make_fixed_width_column(
      data_type{type_id::INT32}, merge_values.size(), mask_state::UNALLOCATED);
    thrust::fill(rmm::exec_policy(cudf::get_default_stream()),
                 merge_keys->mutable_view().template begin<int>(),
                 merge_keys->mutable_view().template end<int>(),
                 0);

    cudf::table_view key_table({*merge_keys});
    cudf::groupby::groupby gb(key_table);
    std::vector<cudf::groupby::aggregation_request> requests;
    std::vector<std::unique_ptr<cudf::groupby_aggregation>> aggregations;
    aggregations.push_back(
      cudf::make_merge_tdigest_aggregation<cudf::groupby_aggregation>(merge_delta));
    requests.push_back({merge_values, std::move(aggregations)});
    auto result = gb.aggregate(requests);
    return std::move(result.second[0].results[0]);
  }
};

template <typename T>
struct TDigestAllTypes : public cudf::test::BaseFixture {
};
TYPED_TEST_SUITE(TDigestAllTypes, cudf::test::NumericTypes);

TYPED_TEST(TDigestAllTypes, Simple)
{
  using T = TypeParam;
  tdigest_simple_aggregation<T>(tdigest_groupby_simple_op{});
}

TYPED_TEST(TDigestAllTypes, SimpleWithNulls)
{
  using T = TypeParam;
  tdigest_simple_with_nulls_aggregation<T>(tdigest_groupby_simple_op{});
}

TYPED_TEST(TDigestAllTypes, AllNull)
{
  using T = TypeParam;
  tdigest_simple_all_nulls_aggregation<T>(tdigest_groupby_simple_op{});
}

TYPED_TEST(TDigestAllTypes, LargeGroups)
{
  auto _values    = generate_standardized_percentile_distribution(data_type{type_id::FLOAT64});
  int const delta = 1000;

  // generate a random set of keys
  std::vector<int> h_keys;
  h_keys.reserve(_values->size());
  auto iter = thrust::make_counting_iterator(0);
  std::transform(iter, iter + _values->size(), std::back_inserter(h_keys), [](int i) {
    return static_cast<int>(round(rand_range(0, 8)));
  });
  cudf::test::fixed_width_column_wrapper<int> _keys(h_keys.begin(), h_keys.end());

  // group the input values together
  cudf::table_view k({_keys});
  cudf::groupby::groupby setup_gb(k);
  cudf::table_view v({*_values});
  auto groups = setup_gb.get_groups(v);

  // slice it all up so we have keys/columns for everything.
  std::vector<column_view> keys;
  std::vector<column_view> values;
  for (size_t idx = 0; idx < groups.offsets.size() - 1; idx++) {
    auto k =
      cudf::slice(groups.keys->get_column(0), {groups.offsets[idx], groups.offsets[idx + 1]});
    keys.push_back(k[0]);

    auto v =
      cudf::slice(groups.values->get_column(0), {groups.offsets[idx], groups.offsets[idx + 1]});
    values.push_back(v[0]);
  }

  // generate a separate tdigest for each group
  std::vector<std::unique_ptr<column>> parts;
  std::transform(
    iter, iter + values.size(), std::back_inserter(parts), [&keys, &values, delta](int i) {
      cudf::table_view t({keys[i]});
      cudf::groupby::groupby gb(t);
      std::vector<cudf::groupby::aggregation_request> requests;
      std::vector<std::unique_ptr<cudf::groupby_aggregation>> aggregations;
      aggregations.push_back(cudf::make_tdigest_aggregation<cudf::groupby_aggregation>(delta));
      requests.push_back({values[i], std::move(aggregations)});
      auto result = gb.aggregate(requests);
      return std::move(result.second[0].results[0]);
    });
  std::vector<column_view> part_views;
  std::transform(parts.begin(),
                 parts.end(),
                 std::back_inserter(part_views),
                 [](std::unique_ptr<column> const& col) { return col->view(); });
  auto merged_parts = cudf::concatenate(part_views);

  // generate a tdigest on the whole input set
  cudf::table_view t({_keys});
  cudf::groupby::groupby gb(t);
  std::vector<cudf::groupby::aggregation_request> requests;
  std::vector<std::unique_ptr<cudf::groupby_aggregation>> aggregations;
  aggregations.push_back(cudf::make_tdigest_aggregation<cudf::groupby_aggregation>(delta));
  requests.push_back({*_values, std::move(aggregations)});
  auto result = gb.aggregate(requests);

  // verify that they end up the same.
  CUDF_TEST_EXPECT_COLUMNS_EQUIVALENT(*result.second[0].results[0], *merged_parts);
}

struct TDigestTest : public cudf::test::BaseFixture {
};

TEST_F(TDigestTest, EmptyMixed)
{
  cudf::test::fixed_width_column_wrapper<double> values{
    {123456.78, 10.0, 20.0, 25.0, 30.0, 40.0, 50.0, 60.0, 70.0}, {1, 0, 0, 1, 0, 0, 1, 1, 0}};
  cudf::test::strings_column_wrapper keys{"b", "a", "c", "c", "d", "d", "e", "e", "f"};

  auto const delta = 1000;
  cudf::table_view t({keys});
  cudf::groupby::groupby gb(t);
  std::vector<cudf::groupby::aggregation_request> requests;
  std::vector<std::unique_ptr<cudf::groupby_aggregation>> aggregations;
  aggregations.push_back(cudf::make_tdigest_aggregation<cudf::groupby_aggregation>(delta));
  requests.push_back({values, std::move(aggregations)});
  auto result = gb.aggregate(requests);

  using FCW     = cudf::test::fixed_width_column_wrapper<double>;
  auto expected = make_expected_tdigest_column({{FCW{}, FCW{}, 0, 0},
                                                {FCW{123456.78}, FCW{1.0}, 123456.78, 123456.78},
                                                {FCW{25.0}, FCW{1.0}, 25.0, 25.0},
                                                {FCW{}, FCW{}, 0, 0},
                                                {FCW{50.0, 60.0}, FCW{1.0, 1.0}, 50.0, 60.0},
                                                {FCW{}, FCW{}, 0, 0}});

  CUDF_TEST_EXPECT_COLUMNS_EQUAL(*result.second[0].results[0], *expected);
}

TEST_F(TDigestTest, LargeInputDouble)
{
  tdigest_simple_large_input_double_aggregation(tdigest_groupby_simple_op{});
}

TEST_F(TDigestTest, LargeInputInt)
{
  tdigest_simple_large_input_int_aggregation(tdigest_groupby_simple_op{});
}

TEST_F(TDigestTest, LargeInputDecimal)
{
  tdigest_simple_large_input_decimal_aggregation(tdigest_groupby_simple_op{});
}

struct TDigestMergeTest : public cudf::test::BaseFixture {
};

// Note: there is no need to test different types here as the internals of a tdigest are always
// the same regardless of input.
TEST_F(TDigestMergeTest, Simple)
{
  tdigest_merge_simple(tdigest_groupby_simple_op{}, tdigest_groupby_simple_merge_op{});
}

struct key_groups {
  __device__ size_type operator()(size_type i) { return i < 250000 ? 0 : 1; }
};
TEST_F(TDigestMergeTest, Grouped)
{
  auto values = generate_standardized_percentile_distribution(data_type{type_id::FLOAT64});
  CUDF_EXPECTS(values->size() == 750000, "Unexpected distribution size");
  // all in the same group
  auto keys = cudf::make_fixed_width_column(
    data_type{type_id::INT32}, values->size(), mask_state::UNALLOCATED);
  // 3 groups. 0-250000 in group 0.  250000-500000 in group 1 and 500000-750000 in group 1
  auto key_iter = cudf::detail::make_counting_transform_iterator(0, key_groups{});
  thrust::copy(rmm::exec_policy(cudf::get_default_stream()),
               key_iter,
               key_iter + keys->size(),
               keys->mutable_view().template begin<int>());

  auto split_values         = cudf::split(*values, {250000, 500000});
  auto grouped_split_values = cudf::split(*values, {250000});
  auto split_keys           = cudf::split(*keys, {250000, 500000});

  int const delta = 1000;

  // generate separate digests
  std::vector<std::unique_ptr<column>> parts;
  auto iter = thrust::make_counting_iterator(0);
  std::transform(
    iter,
    iter + split_values.size(),
    std::back_inserter(parts),
    [&split_keys, &split_values, delta](int i) {
      cudf::table_view t({split_keys[i]});
      cudf::groupby::groupby gb(t);
      std::vector<cudf::groupby::aggregation_request> requests;
      std::vector<std::unique_ptr<cudf::groupby_aggregation>> aggregations;
      aggregations.push_back(cudf::make_tdigest_aggregation<cudf::groupby_aggregation>(delta));
      requests.push_back({split_values[i], std::move(aggregations)});
      auto result = gb.aggregate(requests);
      return std::move(result.second[0].results[0]);
    });
  std::vector<column_view> part_views;
  std::transform(parts.begin(),
                 parts.end(),
                 std::back_inserter(part_views),
                 [](std::unique_ptr<column> const& col) { return col->view(); });

  // merge delta = 1000
  {
    int const merge_delta = 1000;

    // merge them
    auto merge_input = cudf::concatenate(part_views);
    cudf::test::fixed_width_column_wrapper<int> merge_keys{0, 1, 1};
    cudf::table_view key_table({merge_keys});
    cudf::groupby::groupby gb(key_table);
    std::vector<cudf::groupby::aggregation_request> requests;
    std::vector<std::unique_ptr<cudf::groupby_aggregation>> aggregations;
    aggregations.push_back(
      cudf::make_merge_tdigest_aggregation<cudf::groupby_aggregation>(merge_delta));
    requests.push_back({*merge_input, std::move(aggregations)});
    auto result = gb.aggregate(requests);

    CUDF_EXPECTS(result.second[0].results[0]->size() == 2, "Unexpected tdigest merge result size");
    cudf::tdigest::tdigest_column_view tdv(*result.second[0].results[0]);

    // verify centroids
    std::vector<expected_value> expected{// group 0
                                         {0, 0.00013945158577498588, 2},
                                         {10, 0.04804393446447509375, 50},
                                         {66, 2.10089484962640948851, 316},
                                         {139, 8.92977366346101852912, 601},
                                         {243, 23.89152910016953867967, 784},
                                         {366, 41.62636569363655780762, 586},
                                         {432, 47.73085102980330418632, 326},
                                         {460, 49.20637897385523018556, 196},
                                         {501, 49.99998311512171511595, 1},
                                         // group 1
                                         {502 + 0, 50.00022508669655252334, 2},
                                         {502 + 15, 50.05415694538910287292, 74},
                                         {502 + 70, 51.21421484112906341579, 334},
                                         {502 + 150, 55.19367617848146778670, 635},
                                         {502 + 260, 63.24605285552920008740, 783},
                                         {502 + 380, 76.99522005804017510400, 1289},
                                         {502 + 440, 84.22673817294192133431, 758},
                                         {502 + 490, 88.11787981529532487457, 784},
                                         {502 + 555, 93.02766411136053648079, 704},
                                         {502 + 618, 96.91486035315536184953, 516},
                                         {502 + 710, 99.87755861436669135855, 110},
                                         {502 + 733, 99.99970905482754801596, 1}};
    tdigest_sample_compare(tdv, expected);

    // verify min/max
    auto split_results = cudf::split(*result.second[0].results[0], {1});
    auto iter          = thrust::make_counting_iterator(0);
    std::for_each(iter, iter + split_results.size(), [&](size_type i) {
      auto copied = std::make_unique<column>(split_results[i]);
      tdigest_minmax_compare<double>(cudf::tdigest::tdigest_column_view(*copied),
                                     grouped_split_values[i]);
    });
  }

  // merge delta = 100
  {
    int const merge_delta = 100;

    // merge them
    auto merge_input = cudf::concatenate(part_views);
    cudf::test::fixed_width_column_wrapper<int> merge_keys{0, 1, 1};
    cudf::table_view key_table({merge_keys});
    cudf::groupby::groupby gb(key_table);
    std::vector<cudf::groupby::aggregation_request> requests;
    std::vector<std::unique_ptr<cudf::groupby_aggregation>> aggregations;
    aggregations.push_back(
      cudf::make_merge_tdigest_aggregation<cudf::groupby_aggregation>(merge_delta));
    requests.push_back({*merge_input, std::move(aggregations)});
    auto result = gb.aggregate(requests);

    CUDF_EXPECTS(result.second[0].results[0]->size() == 2, "Unexpected tdigest merge result size");
    cudf::tdigest::tdigest_column_view tdv(*result.second[0].results[0]);

    // verify centroids
    std::vector<expected_value> expected{// group 0
                                         {0, 0.02182479870203561656, 231},
                                         {3, 0.60625795002234528219, 1688},
                                         {13, 8.40462931740497687372, 5867},
                                         {27, 28.79997783486397722186, 7757},
                                         {35, 40.22391421196020644402, 6224},
                                         {45, 48.96506331299028857984, 2225},
                                         {50, 49.99979491345574444949, 4},
                                         // group 1
                                         {51 + 0, 50.02171921312970681583, 460},
                                         {51 + 5, 51.45308398121498072442, 5074},
                                         {51 + 11, 55.96880716301625113829, 10011},
                                         {51 + 22, 70.18029861315150697010, 15351},
                                         {51 + 38, 92.65943436519887654867, 10718},
                                         {51 + 47, 99.27745505225347244505, 3639}};
    tdigest_sample_compare(tdv, expected);

    // verify min/max
    auto split_results = cudf::split(*result.second[0].results[0], {1});
    auto iter          = thrust::make_counting_iterator(0);
    std::for_each(iter, iter + split_results.size(), [&](size_type i) {
      auto copied = std::make_unique<column>(split_results[i]);
      tdigest_minmax_compare<double>(cudf::tdigest::tdigest_column_view(*copied),
                                     grouped_split_values[i]);
    });
  }

  // merge delta = 10
  {
    int const merge_delta = 10;

    // merge them
    auto merge_input = cudf::concatenate(part_views);
    cudf::test::fixed_width_column_wrapper<int> merge_keys{0, 1, 1};
    cudf::table_view key_table({merge_keys});
    cudf::groupby::groupby gb(key_table);
    std::vector<cudf::groupby::aggregation_request> requests;
    std::vector<std::unique_ptr<cudf::groupby_aggregation>> aggregations;
    aggregations.push_back(
      cudf::make_merge_tdigest_aggregation<cudf::groupby_aggregation>(merge_delta));
    requests.push_back({*merge_input, std::move(aggregations)});
    auto result = gb.aggregate(requests);

    CUDF_EXPECTS(result.second[0].results[0]->size() == 2, "Unexpected tdigest merge result size");
    cudf::tdigest::tdigest_column_view tdv(*result.second[0].results[0]);

    // verify centroids
    std::vector<expected_value> expected{// group 0
                                         {0, 2.34644806683495144028, 23623},
                                         {1, 10.95523693698660672169, 62290},
                                         {2, 24.90731657803452847588, 77208},
                                         {3, 38.88062495289155862110, 62658},
                                         {4, 47.56288303840698006297, 24217},
                                         {5, 49.99979491345574444949, 4},
                                         // group 1
                                         {6 + 0, 52.40174463129091719793, 47410},
                                         {6 + 1, 60.97025126481504031517, 124564},
                                         {6 + 2, 74.91722742839780835311, 154387},
                                         {6 + 3, 88.87559489177009197647, 124810},
                                         {6 + 4, 97.55823307073454486726, 48817},
                                         {6 + 5, 99.99901807905750672489, 12}};
    tdigest_sample_compare(tdv, expected);

    // verify min/max
    auto split_results = cudf::split(*result.second[0].results[0], {1});
    auto iter          = thrust::make_counting_iterator(0);
    std::for_each(iter, iter + split_results.size(), [&](size_type i) {
      auto copied = std::make_unique<column>(split_results[i]);
      tdigest_minmax_compare<double>(cudf::tdigest::tdigest_column_view(*copied),
                                     grouped_split_values[i]);
    });
  }
}

TEST_F(TDigestMergeTest, Empty) { tdigest_merge_empty(tdigest_groupby_simple_merge_op{}); }

TEST_F(TDigestMergeTest, EmptyGroups)
{
  cudf::test::fixed_width_column_wrapper<double> values_b{{126, 15, 1, 99, 67, 55, 2},
                                                          {1, 0, 0, 1, 1, 1, 1}};
  cudf::test::fixed_width_column_wrapper<double> values_d{{100, 200, 300, 400, 500, 600, 700},
                                                          {1, 1, 1, 1, 1, 1, 0}};
  cudf::test::fixed_width_column_wrapper<int> keys{0, 0, 0, 0, 0, 0, 0};
  int const delta = 1000;

  auto a = cudf::tdigest::detail::make_empty_tdigest_column(cudf::get_default_stream());
  auto b = cudf::type_dispatcher(
    static_cast<column_view>(values_b).type(), tdigest_gen_grouped{}, keys, values_b, delta);
  auto c = cudf::tdigest::detail::make_empty_tdigest_column(cudf::get_default_stream());
  auto d = cudf::type_dispatcher(
    static_cast<column_view>(values_d).type(), tdigest_gen_grouped{}, keys, values_d, delta);
  auto e = cudf::tdigest::detail::make_empty_tdigest_column(cudf::get_default_stream());

  std::vector<column_view> cols;
  cols.push_back(*a);
  cols.push_back(*b);
  cols.push_back(*c);
  cols.push_back(*d);
  cols.push_back(*e);
  auto values = cudf::concatenate(cols);

  cudf::test::fixed_width_column_wrapper<int> merge_keys{0, 0, 1, 0, 2};

  cudf::table_view t({merge_keys});
  cudf::groupby::groupby gb(t);
  std::vector<cudf::groupby::aggregation_request> requests;
  std::vector<std::unique_ptr<cudf::groupby_aggregation>> aggregations;
  aggregations.push_back(cudf::make_merge_tdigest_aggregation<cudf::groupby_aggregation>(delta));
  requests.push_back({*values, std::move(aggregations)});
  auto result = gb.aggregate(requests);

  using FCW = cudf::test::fixed_width_column_wrapper<double>;
  cudf::test::fixed_width_column_wrapper<double> expected_means{
    2, 55, 67, 99, 100, 126, 200, 300, 400, 500, 600};
  cudf::test::fixed_width_column_wrapper<double> expected_weights{1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1};
  auto expected = make_expected_tdigest_column(
    {{expected_means, expected_weights, 2, 600}, {FCW{}, FCW{}, 0, 0}, {FCW{}, FCW{}, 0, 0}});

  CUDF_TEST_EXPECT_COLUMNS_EQUAL(*expected, *result.second[0].results[0]);
}

}  // namespace test
}  // namespace cudf
