#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2021, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "arrow/util/tdigest.h"

#include <cudf/detail/aggregation/aggregation.hpp>
#include <cudf/detail/tdigest/tdigest.hpp>
#include <cudf/lists/lists_column_view.hpp>
#include <cudf/tdigest/tdigest_column_view.cuh>

#include <cudf_test/base_fixture.hpp>
#include <cudf_test/column_wrapper.hpp>
#include <cudf_test/iterator_utilities.hpp>
#include <cudf_test/type_lists.hpp>

#include <rmm/exec_policy.hpp>

#include <tests/groupby/groupby_test_util.hpp>

#include <thrust/fill.h>

namespace cudf {
namespace test {

using namespace cudf;
using expected_value = thrust::tuple<size_type, double, double>;

template <typename T>
struct TDigestAllTypes : public cudf::test::BaseFixture {
};
TYPED_TEST_SUITE(TDigestAllTypes, cudf::test::NumericTypes);

template <typename T>
struct column_min {
  __device__ double operator()(device_span<T const> vals)
  {
    return static_cast<double>(*thrust::min_element(thrust::seq, vals.begin(), vals.end()));
  }
};

template <typename T>
struct column_max {
  __device__ double operator()(device_span<T const> vals)
  {
    return static_cast<double>(*thrust::max_element(thrust::seq, vals.begin(), vals.end()));
  }
};

struct tdigest_gen {
  template <
    typename T,
    typename std::enable_if_t<cudf::is_numeric<T>() || cudf::is_fixed_point<T>()>* = nullptr>
  std::unique_ptr<column> operator()(column_view const& keys, column_view const& values, int delta)
  {
    cudf::table_view t({keys});
    cudf::groupby::groupby gb(t);
    std::vector<cudf::groupby::aggregation_request> requests;
    std::vector<std::unique_ptr<cudf::groupby_aggregation>> aggregations;
    aggregations.push_back(cudf::make_tdigest_aggregation<cudf::groupby_aggregation>(delta));
    requests.push_back({values, std::move(aggregations)});
    auto result = gb.aggregate(requests);
    return std::move(result.second[0].results[0]);
  }

  template <
    typename T,
    typename std::enable_if_t<!cudf::is_numeric<T>() && !cudf::is_fixed_point<T>()>* = nullptr>
  std::unique_ptr<column> operator()(column_view const& keys, column_view const& values, int delta)
  {
    CUDF_FAIL("Invalid tdigest test type");
  }
};

void tdigest_sample_compare(cudf::tdigest::tdigest_column_view const& tdv,
                            std::vector<expected_value> const& h_expected)
{
  column_view result_mean   = tdv.means();
  column_view result_weight = tdv.weights();

  auto expected_mean = cudf::make_fixed_width_column(
    data_type{type_id::FLOAT64}, h_expected.size(), mask_state::UNALLOCATED);
  auto expected_weight = cudf::make_fixed_width_column(
    data_type{type_id::FLOAT64}, h_expected.size(), mask_state::UNALLOCATED);
  auto sampled_result_mean = cudf::make_fixed_width_column(
    data_type{type_id::FLOAT64}, h_expected.size(), mask_state::UNALLOCATED);
  auto sampled_result_weight = cudf::make_fixed_width_column(
    data_type{type_id::FLOAT64}, h_expected.size(), mask_state::UNALLOCATED);

  rmm::device_vector<expected_value> expected(h_expected.begin(), h_expected.end());
  auto iter = thrust::make_counting_iterator(0);
  thrust::for_each(
    rmm::exec_policy(rmm::cuda_stream_default),
    iter,
    iter + expected.size(),
    [expected            = expected.data().get(),
     expected_mean       = expected_mean->mutable_view().begin<double>(),
     expected_weight     = expected_weight->mutable_view().begin<double>(),
     result_mean         = result_mean.begin<double>(),
     result_weight       = result_weight.begin<double>(),
     sampled_result_mean = sampled_result_mean->mutable_view().begin<double>(),
     sampled_result_weight =
       sampled_result_weight->mutable_view().begin<double>()] __device__(size_type index) {
      expected_mean[index]         = thrust::get<1>(expected[index]);
      expected_weight[index]       = thrust::get<2>(expected[index]);
      auto const src_index         = thrust::get<0>(expected[index]);
      sampled_result_mean[index]   = result_mean[src_index];
      sampled_result_weight[index] = result_weight[src_index];
    });

  CUDF_TEST_EXPECT_COLUMNS_EQUIVALENT(*expected_mean, *sampled_result_mean);
  CUDF_TEST_EXPECT_COLUMNS_EQUAL(*expected_weight, *sampled_result_weight);
}

template <typename T>
void tdigest_minmax_compare(cudf::tdigest::tdigest_column_view const& tdv,
                            column_view const& input_values)
{
  // verify min/max
  thrust::host_vector<device_span<T const>> h_spans;
  h_spans.push_back({input_values.begin<T>(), static_cast<size_t>(input_values.size())});
  thrust::device_vector<device_span<T const>> spans(h_spans);

  auto expected_min = cudf::make_fixed_width_column(
    data_type{type_id::FLOAT64}, spans.size(), mask_state::UNALLOCATED);
  thrust::transform(rmm::exec_policy(rmm::cuda_stream_default),
                    spans.begin(),
                    spans.end(),
                    expected_min->mutable_view().template begin<double>(),
                    column_min<T>{});
  column_view result_min(data_type{type_id::FLOAT64}, tdv.size(), tdv.min_begin());
  CUDF_TEST_EXPECT_COLUMNS_EQUAL(result_min, *expected_min);

  auto expected_max = cudf::make_fixed_width_column(
    data_type{type_id::FLOAT64}, spans.size(), mask_state::UNALLOCATED);
  thrust::transform(rmm::exec_policy(rmm::cuda_stream_default),
                    spans.begin(),
                    spans.end(),
                    expected_max->mutable_view().template begin<double>(),
                    column_max<T>{});
  column_view result_max(data_type{type_id::FLOAT64}, tdv.size(), tdv.max_begin());
  CUDF_TEST_EXPECT_COLUMNS_EQUAL(result_max, *expected_max);
}

struct expected_tdigest {
  column_view mean;
  column_view weight;
  double min, max;
};

std::unique_ptr<column> make_expected_tdigest_column(std::vector<expected_tdigest> const& groups)
{
  std::vector<std::unique_ptr<column>> tdigests;

  // make an individual digest
  auto make_digest = [&](expected_tdigest const& tdigest) {
    std::vector<std::unique_ptr<column>> inner_children;
    inner_children.push_back(std::make_unique<cudf::column>(tdigest.mean));
    inner_children.push_back(std::make_unique<cudf::column>(tdigest.weight));
    // tdigest struct
    auto tdigests =
      cudf::make_structs_column(tdigest.mean.size(), std::move(inner_children), 0, {});

    std::vector<offset_type> h_offsets{0, tdigest.mean.size()};
    auto offsets =
      cudf::make_fixed_width_column(data_type{type_id::INT32}, 2, mask_state::UNALLOCATED);
    hipMemcpy(offsets->mutable_view().begin<offset_type>(),
               h_offsets.data(),
               sizeof(offset_type) * 2,
               hipMemcpyHostToDevice);

    auto list = cudf::make_lists_column(1, std::move(offsets), std::move(tdigests), 0, {});

    auto min_col =
      cudf::make_fixed_width_column(data_type{type_id::FLOAT64}, 1, mask_state::UNALLOCATED);
    thrust::fill(rmm::exec_policy(rmm::cuda_stream_default),
                 min_col->mutable_view().begin<double>(),
                 min_col->mutable_view().end<double>(),
                 tdigest.min);
    auto max_col =
      cudf::make_fixed_width_column(data_type{type_id::FLOAT64}, 1, mask_state::UNALLOCATED);
    thrust::fill(rmm::exec_policy(rmm::cuda_stream_default),
                 max_col->mutable_view().begin<double>(),
                 max_col->mutable_view().end<double>(),
                 tdigest.max);

    std::vector<std::unique_ptr<column>> children;
    children.push_back(std::move(list));
    children.push_back(std::move(min_col));
    children.push_back(std::move(max_col));
    return make_structs_column(1, std::move(children), 0, {});
  };

  // build the individual digests
  std::transform(groups.begin(), groups.end(), std::back_inserter(tdigests), make_digest);

  // concatenate them
  std::vector<column_view> views;
  std::transform(tdigests.begin(),
                 tdigests.end(),
                 std::back_inserter(views),
                 [](std::unique_ptr<column> const& c) { return c->view(); });

  return cudf::concatenate(views);
}

TYPED_TEST(TDigestAllTypes, Simple)
{
  using T = TypeParam;

  // create a tdigest that has far fewer values in it than the delta value. this should result
  // in every value remaining uncompressed
  cudf::test::fixed_width_column_wrapper<T> values{126, 15, 1, 99, 67};
  cudf::test::fixed_width_column_wrapper<int> keys{0, 0, 0, 0, 0};
  int const delta = 1000;
  auto result     = cudf::type_dispatcher(
    static_cast<column_view>(values).type(), tdigest_gen{}, keys, values, delta);

  cudf::test::fixed_width_column_wrapper<T> raw_mean({1, 15, 67, 99, 126});
  cudf::test::fixed_width_column_wrapper<double> weight{1, 1, 1, 1, 1};
  auto mean        = cudf::cast(raw_mean, data_type{type_id::FLOAT64});
  double const min = 1;
  double const max = 126;
  auto expected    = make_expected_tdigest_column({{*mean,
                                                 weight,
                                                 static_cast<double>(static_cast<T>(min)),
                                                 static_cast<double>(static_cast<T>(max))}});

  CUDF_TEST_EXPECT_COLUMNS_EQUAL(*result, *expected);
}

TYPED_TEST(TDigestAllTypes, SimpleWithNulls)
{
  using T = TypeParam;

  // create a tdigest that has far fewer values in it than the delta value. this should result
  // in every value remaining uncompressed
  cudf::test::fixed_width_column_wrapper<T> values{{122, 15, 1, 99, 67, 101, 100, 84, 44, 2},
                                                   {1, 0, 1, 0, 1, 0, 1, 0, 1, 0}};
  cudf::test::fixed_width_column_wrapper<int> keys{0, 0, 0, 0, 0, 0, 0, 0, 0, 0};
  int const delta = 1000;
  auto result     = cudf::type_dispatcher(
    static_cast<column_view>(values).type(), tdigest_gen{}, keys, values, delta);

  cudf::test::fixed_width_column_wrapper<T> raw_mean({1, 44, 67, 100, 122});
  cudf::test::fixed_width_column_wrapper<double> weight{1, 1, 1, 1, 1};
  auto mean        = cudf::cast(raw_mean, data_type{type_id::FLOAT64});
  double const min = 1;
  double const max = 122;
  auto expected    = make_expected_tdigest_column({{*mean,
                                                 weight,
                                                 static_cast<double>(static_cast<T>(min)),
                                                 static_cast<double>(static_cast<T>(max))}});

  CUDF_TEST_EXPECT_COLUMNS_EQUAL(*result, *expected);
}

TYPED_TEST(TDigestAllTypes, AllNull)
{
  using T = TypeParam;

  // create a tdigest that has far fewer values in it than the delta value. this should result
  // in every value remaining uncompressed
  cudf::test::fixed_width_column_wrapper<T> values{{122, 15, 1, 99, 67, 101, 100, 84, 44, 2},
                                                   {0, 0, 0, 0, 0, 0, 0, 0, 0, 0}};
  cudf::test::fixed_width_column_wrapper<int> keys{0, 0, 0, 0, 0, 0, 0, 0, 0, 0};
  int const delta = 1000;
  auto result     = cudf::type_dispatcher(
    static_cast<column_view>(values).type(), tdigest_gen{}, keys, values, delta);

  // NOTE: an empty tdigest column still has 1 row.
  auto expected = cudf::detail::tdigest::make_empty_tdigest_column();

  CUDF_TEST_EXPECT_COLUMNS_EQUAL(*result, *expected);
}

TYPED_TEST(TDigestAllTypes, LargeGroups)
{
  auto _values    = generate_standardized_percentile_distribution(data_type{type_id::FLOAT64});
  int const delta = 1000;

  // generate a random set of keys
  std::vector<int> h_keys;
  h_keys.reserve(_values->size());
  auto iter = thrust::make_counting_iterator(0);
  std::transform(iter, iter + _values->size(), std::back_inserter(h_keys), [](int i) {
    return static_cast<int>(round(rand_range(0, 8)));
  });
  cudf::test::fixed_width_column_wrapper<int> _keys(h_keys.begin(), h_keys.end());

  // group the input values together
  cudf::table_view k({_keys});
  cudf::groupby::groupby setup_gb(k);
  cudf::table_view v({*_values});
  auto groups = setup_gb.get_groups(v);

  // slice it all up so we have keys/columns for everything.
  std::vector<column_view> keys;
  std::vector<column_view> values;
  for (size_t idx = 0; idx < groups.offsets.size() - 1; idx++) {
    auto k =
      cudf::slice(groups.keys->get_column(0), {groups.offsets[idx], groups.offsets[idx + 1]});
    keys.push_back(k[0]);

    auto v =
      cudf::slice(groups.values->get_column(0), {groups.offsets[idx], groups.offsets[idx + 1]});
    values.push_back(v[0]);
  }

  // generate a seperate tdigest for each group
  std::vector<std::unique_ptr<column>> parts;
  std::transform(
    iter, iter + values.size(), std::back_inserter(parts), [&keys, &values, delta](int i) {
      cudf::table_view t({keys[i]});
      cudf::groupby::groupby gb(t);
      std::vector<cudf::groupby::aggregation_request> requests;
      std::vector<std::unique_ptr<cudf::groupby_aggregation>> aggregations;
      aggregations.push_back(cudf::make_tdigest_aggregation<cudf::groupby_aggregation>(delta));
      requests.push_back({values[i], std::move(aggregations)});
      auto result = gb.aggregate(requests);
      return std::move(result.second[0].results[0]);
    });
  std::vector<column_view> part_views;
  std::transform(parts.begin(),
                 parts.end(),
                 std::back_inserter(part_views),
                 [](std::unique_ptr<column> const& col) { return col->view(); });
  auto merged_parts = cudf::concatenate(part_views);

  // generate a tdigest on the whole input set
  cudf::table_view t({_keys});
  cudf::groupby::groupby gb(t);
  std::vector<cudf::groupby::aggregation_request> requests;
  std::vector<std::unique_ptr<cudf::groupby_aggregation>> aggregations;
  aggregations.push_back(cudf::make_tdigest_aggregation<cudf::groupby_aggregation>(delta));
  requests.push_back({*_values, std::move(aggregations)});
  auto result = gb.aggregate(requests);

  // verify that they end up the same.
  CUDF_TEST_EXPECT_COLUMNS_EQUIVALENT(*result.second[0].results[0], *merged_parts);
}

struct TDigestTest : public cudf::test::BaseFixture {
};

TEST_F(TDigestTest, EmptyMixed)
{
  cudf::test::fixed_width_column_wrapper<double> values{
    {123456.78, 10.0, 20.0, 25.0, 30.0, 40.0, 50.0, 60.0, 70.0}, {1, 0, 0, 1, 0, 0, 1, 1, 0}};
  cudf::test::strings_column_wrapper keys{"b", "a", "c", "c", "d", "d", "e", "e", "f"};

  auto const delta = 1000;
  cudf::table_view t({keys});
  cudf::groupby::groupby gb(t);
  std::vector<cudf::groupby::aggregation_request> requests;
  std::vector<std::unique_ptr<cudf::groupby_aggregation>> aggregations;
  aggregations.push_back(cudf::make_tdigest_aggregation<cudf::groupby_aggregation>(delta));
  requests.push_back({values, std::move(aggregations)});
  auto result = gb.aggregate(requests);

  using FCW     = cudf::test::fixed_width_column_wrapper<double>;
  auto expected = make_expected_tdigest_column({{FCW{}, FCW{}, 0, 0},
                                                {FCW{123456.78}, FCW{1.0}, 123456.78, 123456.78},
                                                {FCW{25.0}, FCW{1.0}, 25.0, 25.0},
                                                {FCW{}, FCW{}, 0, 0},
                                                {FCW{50.0, 60.0}, FCW{1.0, 1.0}, 50.0, 60.0},
                                                {FCW{}, FCW{}, 0, 0}});

  CUDF_TEST_EXPECT_COLUMNS_EQUAL(*result.second[0].results[0], *expected);
}

TEST_F(TDigestTest, LargeInputDouble)
{
  // these tests are being done explicitly because of the way we have to precompute the correct
  // answers. since the input values generated by the generate_distribution() function below are
  // cast to specific types -before- being sent into the aggregation, I can't (safely) just use the
  // expected values that you get when using doubles all the way through.  so I have to pregenerate
  // the correct answers for each type by hand. so, we'll choose a reasonable subset (double,
  // decimal, int, bool)

  auto values = generate_standardized_percentile_distribution(data_type{type_id::FLOAT64});
  // all in the same group
  auto keys = cudf::make_fixed_width_column(
    data_type{type_id::INT32}, values->size(), mask_state::UNALLOCATED);
  thrust::fill(rmm::exec_policy(rmm::cuda_stream_default),
               keys->mutable_view().template begin<int>(),
               keys->mutable_view().template end<int>(),
               0);

  // compare against a sample of known/expected values (which themselves were verified against the
  // Arrow implementation)

  // delta 1000
  {
    int const delta = 1000;
    auto result =
      cudf::type_dispatcher(values->view().type(), tdigest_gen{}, *keys, *values, delta);
    std::vector<expected_value> expected{{0, 0.00040692343794663995, 7},
                                         {10, 0.16234555627091204477, 153},
                                         {59, 5.12764811246045937310, 858},
                                         {250, 62.54581814492237157310, 2356},
                                         {368, 87.85834376680742252574, 1735},
                                         {409, 94.07685720279611985006, 1272},
                                         {491, 99.94197663121231300920, 130},
                                         {500, 99.99969880795092080916, 2}};
    cudf::tdigest::tdigest_column_view tdv(*result);

    tdigest_sample_compare(tdv, expected);

    // verify min/max
    tdigest_minmax_compare<double>(tdv, *values);
  }

  // delta 100
  {
    int const delta = 100;
    auto result =
      cudf::type_dispatcher(values->view().type(), tdigest_gen{}, *keys, *values, delta);
    std::vector<expected_value> expected{{0, 0.07265722021410986331, 739},
                                         {7, 8.19766194442652640362, 10693},
                                         {16, 36.82277869518204482802, 20276},
                                         {29, 72.95424834129075009059, 22623},
                                         {38, 90.61229683516096145013, 15581},
                                         {46, 99.07283498858802772702, 5142},
                                         {50, 99.99970905482754801596, 1}};
    cudf::tdigest::tdigest_column_view tdv(*result);

    tdigest_sample_compare(tdv, expected);

    // verify min/max
    tdigest_minmax_compare<double>(tdv, *values);
  }

  // delta 10
  {
    int const delta = 10;
    auto result =
      cudf::type_dispatcher(values->view().type(), tdigest_gen{}, *keys, *values, delta);
    std::vector<expected_value> expected{{0, 7.15508346777729631327, 71618},
                                         {1, 33.04971680740474226923, 187499},
                                         {2, 62.50566666553867634093, 231762},
                                         {3, 83.46216572053654658703, 187500},
                                         {4, 96.42204425201593664951, 71620},
                                         {5, 99.99970905482754801596, 1}};
    cudf::tdigest::tdigest_column_view tdv(*result);

    tdigest_sample_compare(tdv, expected);

    // verify min/max
    tdigest_minmax_compare<double>(tdv, *values);
  }
}

TEST_F(TDigestTest, LargeInputInt)
{
  // these tests are being done explicitly because of the way we have to precompute the correct
  // answers. since the input values generated by the generate_distribution() function below are
  // cast to specific types -before- being sent into the aggregation, I can't (safely) just use the
  // expected values that you get when using doubles all the way through.  so I have to pregenerate
  // the correct answers for each type by hand. so, we'll choose a reasonable subset (double,
  // decimal, int, bool)

  auto values = generate_standardized_percentile_distribution(data_type{type_id::INT32});
  // all in the same group
  auto keys = cudf::make_fixed_width_column(
    data_type{type_id::INT32}, values->size(), mask_state::UNALLOCATED);
  thrust::fill(rmm::exec_policy(rmm::cuda_stream_default),
               keys->mutable_view().template begin<int>(),
               keys->mutable_view().template end<int>(),
               0);

  // compare against a sample of known/expected values (which themselves were verified against the
  // Arrow implementation)

  // delta 1000
  {
    int const delta = 1000;
    auto result =
      cudf::type_dispatcher(values->view().type(), tdigest_gen{}, *keys, *values, delta);
    std::vector<expected_value> expected{{0, 0, 7},
                                         {14, 0, 212},
                                         {26, 0.83247422680412408447, 388},
                                         {44, 2, 648},
                                         {45, 2.42598187311178170589, 662},
                                         {342, 82.75190258751908345403, 1971},
                                         {383, 90, 1577},
                                         {417, 94.88376068376066996279, 1170},
                                         {418, 95, 1157},
                                         {479, 99, 307},
                                         {500, 99, 2}};
    cudf::tdigest::tdigest_column_view tdv(*result);

    tdigest_sample_compare(tdv, expected);

    // verify min/max
    tdigest_minmax_compare<int>(tdv, *values);
  }

  // delta 100
  {
    int const delta = 100;
    auto result =
      cudf::type_dispatcher(values->view().type(), tdigest_gen{}, *keys, *values, delta);
    std::vector<expected_value> expected{{0, 0, 739},
                                         {7, 7.71486018890863167741, 10693},
                                         {16, 36.32491615703294485229, 20276},
                                         {29, 72.44392874508245938614, 22623},
                                         {38, 90.14209614273795523332, 15581},
                                         {46, 98.64041229093737683797, 5142},
                                         {50, 99, 1}};
    cudf::tdigest::tdigest_column_view tdv(*result);

    tdigest_sample_compare(tdv, expected);

    // verify min/max
    tdigest_minmax_compare<int>(tdv, *values);
  }

  // delta 10
  {
    int const delta = 10;
    auto result =
      cudf::type_dispatcher(values->view().type(), tdigest_gen{}, *keys, *values, delta);
    std::vector<expected_value> expected{{0, 6.66025300902007799664, 71618},
                                         {1, 32.54912826201739051157, 187499},
                                         {2, 62.00734805533262772315, 231762},
                                         {3, 82.96355733333332693746, 187500},
                                         {4, 95.91280368612116546956, 71620},
                                         {5, 99, 1}};
    cudf::tdigest::tdigest_column_view tdv(*result);

    tdigest_sample_compare(tdv, expected);

    // verify min/max
    tdigest_minmax_compare<int>(tdv, *values);
  }
}

TEST_F(TDigestTest, LargeInputDecimal)
{
  // these tests are being done explicitly because of the way we have to precompute the correct
  // answers. since the input values generated by the generate_distribution() function below are
  // cast to specific types -before- being sent into the aggregation, I can't (safely) just use the
  // expected values that you get when using doubles all the way through.  so I have to pregenerate
  // the correct answers for each type by hand. so, we'll choose a reasonable subset (double,
  // decimal, int, bool)

  auto values = generate_standardized_percentile_distribution(data_type{type_id::DECIMAL32, -4});
  auto cast_values = cudf::cast(*values, data_type{type_id::FLOAT64});
  // all in the same group
  auto keys = cudf::make_fixed_width_column(
    data_type{type_id::INT32}, values->size(), mask_state::UNALLOCATED);
  thrust::fill(rmm::exec_policy(rmm::cuda_stream_default),
               keys->mutable_view().template begin<int>(),
               keys->mutable_view().template end<int>(),
               0);

  // compare against a sample of known/expected values (which themselves were verified against the
  // Arrow implementation)

  // delta 1000
  {
    int const delta = 1000;
    auto result =
      cudf::type_dispatcher(values->view().type(), tdigest_gen{}, *keys, *values, delta);
    std::vector<expected_value> expected{{0, 0.00035714285714285709, 7},
                                         {10, 0.16229738562091505782, 153},
                                         {59, 5.12759696969697031932, 858},
                                         {250, 62.54576854838715860296, 2356},
                                         {368, 87.85829446685879418055, 1735},
                                         {409, 94.07680636792450457051, 1272},
                                         {491, 99.94192461538463589932, 130},
                                         {500, 99.99965000000000259206, 2}};
    cudf::tdigest::tdigest_column_view tdv(*result);

    tdigest_sample_compare(tdv, expected);

    // verify min/max
    tdigest_minmax_compare<double>(tdv, *cast_values);
  }

  // delta 100
  {
    int const delta = 100;
    auto result =
      cudf::type_dispatcher(values->view().type(), tdigest_gen{}, *keys, *values, delta);
    std::vector<expected_value> expected{{0, 0.07260811907983763525, 739},
                                         {7, 8.19761183016926864298, 10693},
                                         {16, 36.82272891595975750079, 20276},
                                         {29, 72.95419827167043536065, 22623},
                                         {38, 90.61224673640975879607, 15581},
                                         {46, 99.07278498638662256326, 5142},
                                         {50, 99.99970000000000425189, 1}};
    cudf::tdigest::tdigest_column_view tdv(*result);

    tdigest_sample_compare(tdv, expected);

    // verify min/max
    tdigest_minmax_compare<double>(tdv, *cast_values);
  }

  // delta 10
  {
    int const delta = 10;
    auto result =
      cudf::type_dispatcher(values->view().type(), tdigest_gen{}, *keys, *values, delta);
    std::vector<expected_value> expected{{0, 7.15503361864335740705, 71618},
                                         {1, 33.04966679715625588187, 187499},
                                         {2, 62.50561666407782013266, 231762},
                                         {3, 83.46211575573336460820, 187500},
                                         {4, 96.42199425300195514410, 71620},
                                         {5, 99.99970000000000425189, 1}};
    cudf::tdigest::tdigest_column_view tdv(*result);

    tdigest_sample_compare(tdv, expected);

    // verify min/max
    tdigest_minmax_compare<double>(tdv, *cast_values);
  }
}

struct TDigestMergeTest : public cudf::test::BaseFixture {
};

// Note: there is no need to test different types here as the internals of a tdigest are always
// the same regardless of input.
TEST_F(TDigestMergeTest, Simple)
{
  auto values = generate_standardized_percentile_distribution(data_type{type_id::FLOAT64});
  CUDF_EXPECTS(values->size() == 750000, "Unexpected distribution size");
  // all in the same group
  auto keys = cudf::make_fixed_width_column(
    data_type{type_id::INT32}, values->size(), mask_state::UNALLOCATED);
  thrust::fill(rmm::exec_policy(rmm::cuda_stream_default),
               keys->mutable_view().template begin<int>(),
               keys->mutable_view().template end<int>(),
               0);

  auto split_values = cudf::split(*values, {250000, 500000});
  auto split_keys   = cudf::split(*keys, {250000, 500000});

  int const delta = 1000;

  // generate seperate digests
  std::vector<std::unique_ptr<column>> parts;
  auto iter = thrust::make_counting_iterator(0);
  std::transform(
    iter,
    iter + split_values.size(),
    std::back_inserter(parts),
    [&split_keys, &split_values, delta](int i) {
      cudf::table_view t({split_keys[i]});
      cudf::groupby::groupby gb(t);
      std::vector<cudf::groupby::aggregation_request> requests;
      std::vector<std::unique_ptr<cudf::groupby_aggregation>> aggregations;
      aggregations.push_back(cudf::make_tdigest_aggregation<cudf::groupby_aggregation>(delta));
      requests.push_back({split_values[i], std::move(aggregations)});
      auto result = gb.aggregate(requests);
      return std::move(result.second[0].results[0]);
    });
  std::vector<column_view> part_views;
  std::transform(parts.begin(),
                 parts.end(),
                 std::back_inserter(part_views),
                 [](std::unique_ptr<column> const& col) { return col->view(); });

  // merge delta = 1000
  {
    int const merge_delta = 1000;

    // merge them
    auto merge_input = cudf::concatenate(part_views);
    cudf::test::fixed_width_column_wrapper<int> merge_keys{0, 0, 0};
    cudf::table_view key_table({merge_keys});
    cudf::groupby::groupby gb(key_table);
    std::vector<cudf::groupby::aggregation_request> requests;
    std::vector<std::unique_ptr<cudf::groupby_aggregation>> aggregations;
    aggregations.push_back(
      cudf::make_merge_tdigest_aggregation<cudf::groupby_aggregation>(merge_delta));
    requests.push_back({*merge_input, std::move(aggregations)});
    auto result = gb.aggregate(requests);

    cudf::tdigest::tdigest_column_view tdv(*result.second[0].results[0]);

    // verify centroids
    std::vector<expected_value> expected{{0, 0.00013945158577498588, 2},
                                         {10, 0.04804393446447510763, 50},
                                         {59, 1.68846964439246893797, 284},
                                         {250, 33.36323141295877547918, 1479},
                                         {368, 65.36307727957283475462, 2292},
                                         {409, 73.95399208218296394080, 1784},
                                         {490, 87.67566167909056673579, 1570},
                                         {491, 87.83119717763385381204, 1570},
                                         {500, 89.24891838334393412424, 1555},
                                         {578, 95.87182997389099625707, 583},
                                         {625, 98.20470345147104751504, 405},
                                         {700, 99.96818381983835877236, 56},
                                         {711, 99.99970905482754801596, 1}};
    tdigest_sample_compare(tdv, expected);

    // verify min/max
    tdigest_minmax_compare<double>(tdv, *values);
  }
}

struct key_groups {
  __device__ size_type operator()(size_type i) { return i < 250000 ? 0 : 1; }
};
TEST_F(TDigestMergeTest, Grouped)
{
  auto values = generate_standardized_percentile_distribution(data_type{type_id::FLOAT64});
  CUDF_EXPECTS(values->size() == 750000, "Unexpected distribution size");
  // all in the same group
  auto keys = cudf::make_fixed_width_column(
    data_type{type_id::INT32}, values->size(), mask_state::UNALLOCATED);
  // 3 groups. 0-250000 in group 0.  250000-500000 in group 1 and 500000-750000 in group 1
  auto key_iter = cudf::detail::make_counting_transform_iterator(0, key_groups{});
  thrust::copy(rmm::exec_policy(rmm::cuda_stream_default),
               key_iter,
               key_iter + keys->size(),
               keys->mutable_view().template begin<int>());

  auto split_values         = cudf::split(*values, {250000, 500000});
  auto grouped_split_values = cudf::split(*values, {250000});
  auto split_keys           = cudf::split(*keys, {250000, 500000});

  int const delta = 1000;

  // generate seperate digests
  std::vector<std::unique_ptr<column>> parts;
  auto iter = thrust::make_counting_iterator(0);
  std::transform(
    iter,
    iter + split_values.size(),
    std::back_inserter(parts),
    [&split_keys, &split_values, delta](int i) {
      cudf::table_view t({split_keys[i]});
      cudf::groupby::groupby gb(t);
      std::vector<cudf::groupby::aggregation_request> requests;
      std::vector<std::unique_ptr<cudf::groupby_aggregation>> aggregations;
      aggregations.push_back(cudf::make_tdigest_aggregation<cudf::groupby_aggregation>(delta));
      requests.push_back({split_values[i], std::move(aggregations)});
      auto result = gb.aggregate(requests);
      return std::move(result.second[0].results[0]);
    });
  std::vector<column_view> part_views;
  std::transform(parts.begin(),
                 parts.end(),
                 std::back_inserter(part_views),
                 [](std::unique_ptr<column> const& col) { return col->view(); });

  // merge delta = 1000
  {
    int const merge_delta = 1000;

    // merge them
    auto merge_input = cudf::concatenate(part_views);
    cudf::test::fixed_width_column_wrapper<int> merge_keys{0, 1, 1};
    cudf::table_view key_table({merge_keys});
    cudf::groupby::groupby gb(key_table);
    std::vector<cudf::groupby::aggregation_request> requests;
    std::vector<std::unique_ptr<cudf::groupby_aggregation>> aggregations;
    aggregations.push_back(
      cudf::make_merge_tdigest_aggregation<cudf::groupby_aggregation>(merge_delta));
    requests.push_back({*merge_input, std::move(aggregations)});
    auto result = gb.aggregate(requests);

    CUDF_EXPECTS(result.second[0].results[0]->size() == 2, "Unexpected tdigest merge result size");
    cudf::tdigest::tdigest_column_view tdv(*result.second[0].results[0]);

    // verify centroids
    std::vector<expected_value> expected{// group 0
                                         {0, 0.00013945158577498588, 2},
                                         {10, 0.04804393446447509375, 50},
                                         {66, 2.10089484962640948851, 316},
                                         {139, 8.92977366346101852912, 601},
                                         {243, 23.89152910016953867967, 784},
                                         {366, 41.62636569363655780762, 586},
                                         {432, 47.73085102980330418632, 326},
                                         {460, 49.20637897385523018556, 196},
                                         {501, 49.99998311512171511595, 1},
                                         // group 1
                                         {502 + 0, 50.00022508669655252334, 2},
                                         {502 + 15, 50.05415694538910287292, 74},
                                         {502 + 70, 51.21421484112906341579, 334},
                                         {502 + 150, 55.19367617848146778670, 635},
                                         {502 + 260, 63.24605285552920008740, 783},
                                         {502 + 380, 76.99522005804017510400, 1289},
                                         {502 + 440, 84.22673817294192133431, 758},
                                         {502 + 490, 88.11787981529532487457, 784},
                                         {502 + 555, 93.02766411136053648079, 704},
                                         {502 + 618, 96.91486035315536184953, 516},
                                         {502 + 710, 99.87755861436669135855, 110},
                                         {502 + 733, 99.99970905482754801596, 1}};
    tdigest_sample_compare(tdv, expected);

    // verify min/max
    auto split_results = cudf::split(*result.second[0].results[0], {1});
    auto iter          = thrust::make_counting_iterator(0);
    std::for_each(iter, iter + split_results.size(), [&](size_type i) {
      auto copied = std::make_unique<column>(split_results[i]);
      tdigest_minmax_compare<double>(cudf::tdigest::tdigest_column_view(*copied),
                                     grouped_split_values[i]);
    });
  }

  // merge delta = 100
  {
    int const merge_delta = 100;

    // merge them
    auto merge_input = cudf::concatenate(part_views);
    cudf::test::fixed_width_column_wrapper<int> merge_keys{0, 1, 1};
    cudf::table_view key_table({merge_keys});
    cudf::groupby::groupby gb(key_table);
    std::vector<cudf::groupby::aggregation_request> requests;
    std::vector<std::unique_ptr<cudf::groupby_aggregation>> aggregations;
    aggregations.push_back(
      cudf::make_merge_tdigest_aggregation<cudf::groupby_aggregation>(merge_delta));
    requests.push_back({*merge_input, std::move(aggregations)});
    auto result = gb.aggregate(requests);

    CUDF_EXPECTS(result.second[0].results[0]->size() == 2, "Unexpected tdigest merge result size");
    cudf::tdigest::tdigest_column_view tdv(*result.second[0].results[0]);

    // verify centroids
    std::vector<expected_value> expected{// group 0
                                         {0, 0.02182479870203561656, 231},
                                         {3, 0.60625795002234528219, 1688},
                                         {13, 8.40462931740497687372, 5867},
                                         {27, 28.79997783486397722186, 7757},
                                         {35, 40.22391421196020644402, 6224},
                                         {45, 48.96506331299028857984, 2225},
                                         {50, 49.99979491345574444949, 4},
                                         // group 1
                                         {51 + 0, 50.02171921312970681583, 460},
                                         {51 + 5, 51.45308398121498072442, 5074},
                                         {51 + 11, 55.96880716301625113829, 10011},
                                         {51 + 22, 70.18029861315150697010, 15351},
                                         {51 + 38, 92.65943436519887654867, 10718},
                                         {51 + 47, 99.27745505225347244505, 3639}};
    tdigest_sample_compare(tdv, expected);

    // verify min/max
    auto split_results = cudf::split(*result.second[0].results[0], {1});
    auto iter          = thrust::make_counting_iterator(0);
    std::for_each(iter, iter + split_results.size(), [&](size_type i) {
      auto copied = std::make_unique<column>(split_results[i]);
      tdigest_minmax_compare<double>(cudf::tdigest::tdigest_column_view(*copied),
                                     grouped_split_values[i]);
    });
  }

  // merge delta = 10
  {
    int const merge_delta = 10;

    // merge them
    auto merge_input = cudf::concatenate(part_views);
    cudf::test::fixed_width_column_wrapper<int> merge_keys{0, 1, 1};
    cudf::table_view key_table({merge_keys});
    cudf::groupby::groupby gb(key_table);
    std::vector<cudf::groupby::aggregation_request> requests;
    std::vector<std::unique_ptr<cudf::groupby_aggregation>> aggregations;
    aggregations.push_back(
      cudf::make_merge_tdigest_aggregation<cudf::groupby_aggregation>(merge_delta));
    requests.push_back({*merge_input, std::move(aggregations)});
    auto result = gb.aggregate(requests);

    CUDF_EXPECTS(result.second[0].results[0]->size() == 2, "Unexpected tdigest merge result size");
    cudf::tdigest::tdigest_column_view tdv(*result.second[0].results[0]);

    // verify centroids
    std::vector<expected_value> expected{// group 0
                                         {0, 2.34644806683495144028, 23623},
                                         {1, 10.95523693698660672169, 62290},
                                         {2, 24.90731657803452847588, 77208},
                                         {3, 38.88062495289155862110, 62658},
                                         {4, 47.56288303840698006297, 24217},
                                         {5, 49.99979491345574444949, 4},
                                         // group 1
                                         {6 + 0, 52.40174463129091719793, 47410},
                                         {6 + 1, 60.97025126481504031517, 124564},
                                         {6 + 2, 74.91722742839780835311, 154387},
                                         {6 + 3, 88.87559489177009197647, 124810},
                                         {6 + 4, 97.55823307073454486726, 48817},
                                         {6 + 5, 99.99901807905750672489, 12}};
    tdigest_sample_compare(tdv, expected);

    // verify min/max
    auto split_results = cudf::split(*result.second[0].results[0], {1});
    auto iter          = thrust::make_counting_iterator(0);
    std::for_each(iter, iter + split_results.size(), [&](size_type i) {
      auto copied = std::make_unique<column>(split_results[i]);
      tdigest_minmax_compare<double>(cudf::tdigest::tdigest_column_view(*copied),
                                     grouped_split_values[i]);
    });
  }
}

TEST_F(TDigestMergeTest, Empty)
{
  // 3 empty tdigests all in the same group
  auto a = cudf::detail::tdigest::make_empty_tdigest_column();
  auto b = cudf::detail::tdigest::make_empty_tdigest_column();
  auto c = cudf::detail::tdigest::make_empty_tdigest_column();
  std::vector<column_view> cols;
  cols.push_back(*a);
  cols.push_back(*b);
  cols.push_back(*c);
  auto values = cudf::concatenate(cols);
  cudf::test::fixed_width_column_wrapper<int> keys{0, 0, 0};

  auto const delta = 1000;
  cudf::table_view t({keys});
  cudf::groupby::groupby gb(t);
  std::vector<cudf::groupby::aggregation_request> requests;
  std::vector<std::unique_ptr<cudf::groupby_aggregation>> aggregations;
  aggregations.push_back(cudf::make_merge_tdigest_aggregation<cudf::groupby_aggregation>(delta));
  requests.push_back({*values, std::move(aggregations)});
  auto result = gb.aggregate(requests);

  auto expected = cudf::detail::tdigest::make_empty_tdigest_column();

  CUDF_TEST_EXPECT_COLUMNS_EQUAL(*expected, *result.second[0].results[0]);
}

TEST_F(TDigestMergeTest, EmptyGroups)
{
  cudf::test::fixed_width_column_wrapper<double> values_b{{126, 15, 1, 99, 67, 55, 2},
                                                          {1, 0, 0, 1, 1, 1, 1}};
  cudf::test::fixed_width_column_wrapper<double> values_d{{100, 200, 300, 400, 500, 600, 700},
                                                          {1, 1, 1, 1, 1, 1, 0}};
  cudf::test::fixed_width_column_wrapper<int> keys{0, 0, 0, 0, 0, 0, 0};
  int const delta = 1000;

  auto a = cudf::detail::tdigest::make_empty_tdigest_column();
  auto b = cudf::type_dispatcher(
    static_cast<column_view>(values_b).type(), tdigest_gen{}, keys, values_b, delta);
  auto c = cudf::detail::tdigest::make_empty_tdigest_column();
  auto d = cudf::type_dispatcher(
    static_cast<column_view>(values_d).type(), tdigest_gen{}, keys, values_d, delta);
  auto e = cudf::detail::tdigest::make_empty_tdigest_column();

  std::vector<column_view> cols;
  cols.push_back(*a);
  cols.push_back(*b);
  cols.push_back(*c);
  cols.push_back(*d);
  cols.push_back(*e);
  auto values = cudf::concatenate(cols);

  cudf::test::fixed_width_column_wrapper<int> merge_keys{0, 0, 1, 0, 2};

  cudf::table_view t({merge_keys});
  cudf::groupby::groupby gb(t);
  std::vector<cudf::groupby::aggregation_request> requests;
  std::vector<std::unique_ptr<cudf::groupby_aggregation>> aggregations;
  aggregations.push_back(cudf::make_merge_tdigest_aggregation<cudf::groupby_aggregation>(delta));
  requests.push_back({*values, std::move(aggregations)});
  auto result = gb.aggregate(requests);

  using FCW = cudf::test::fixed_width_column_wrapper<double>;
  cudf::test::fixed_width_column_wrapper<double> expected_means{
    2, 55, 67, 99, 100, 126, 200, 300, 400, 500, 600};
  cudf::test::fixed_width_column_wrapper<double> expected_weights{1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1};
  auto expected = make_expected_tdigest_column(
    {{expected_means, expected_weights, 2, 600}, {FCW{}, FCW{}, 0, 0}, {FCW{}, FCW{}, 0, 0}});

  CUDF_TEST_EXPECT_COLUMNS_EQUAL(*expected, *result.second[0].results[0]);
}

}  // namespace test
}  // namespace cudf
