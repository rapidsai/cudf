/*
 * Copyright (c) 2020-2024, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "io/utilities/hostdevice_vector.hpp"

#include <cudf_test/base_fixture.hpp>
#include <cudf_test/cudf_gtest.hpp>
#include <cudf_test/testing_main.hpp>
#include <cudf_test/type_lists.hpp>

#include <cudf/detail/utilities/vector_factories.hpp>
#include <cudf/utilities/default_stream.hpp>
#include <cudf/utilities/span.hpp>

#include <rmm/device_buffer.hpp>
#include <rmm/device_vector.hpp>

#include <thrust/device_vector.h>
#include <thrust/host_vector.h>

#include <cstddef>
#include <cstring>
#include <string>

using cudf::device_span;
using cudf::host_span;
using cudf::detail::device_2dspan;
using cudf::detail::host_2dspan;
using cudf::detail::hostdevice_2dvector;

template <typename T>
void expect_equivalent(host_span<T> a, host_span<T> b)
{
  EXPECT_EQ(a.size(), b.size());
  EXPECT_EQ(a.data(), b.data());
}

template <typename T>
void expect_equivalent(cudf::detail::hostdevice_span<T> a, cudf::detail::hostdevice_span<T> b)
{
  EXPECT_EQ(a.size(), b.size());
  EXPECT_EQ(a.host_ptr(), b.host_ptr());
}

template <typename Iterator1, typename T>
void expect_match(Iterator1 expected, size_t expected_size, host_span<T> input)
{
  EXPECT_EQ(expected_size, input.size());
  for (size_t i = 0; i < expected_size; i++) {
    EXPECT_EQ(*(expected + i), *(input.begin() + i));
  }
}

template <typename T>
void expect_match(std::string expected, host_span<T> input)
{
  return expect_match(expected.begin(), expected.size(), input);
}

template <typename T>
void expect_match(std::string expected, cudf::detail::hostdevice_span<T> input)
{
  return expect_match(expected.begin(), expected.size(), host_span<T>(input));
}

std::string const hello_world_message = "hello world";
std::vector<char> create_hello_world_message()
{
  return std::vector<char>(hello_world_message.begin(), hello_world_message.end());
}

class SpanTest : public cudf::test::BaseFixture {};

TEST(SpanTest, CanCreateFullSubspan)
{
  auto message            = create_hello_world_message();
  auto const message_span = host_span<char>(message.data(), message.size());

  expect_equivalent(message_span, message_span.subspan(0, message_span.size()));
}

TEST(SpanTest, CanTakeFirst)
{
  auto message            = create_hello_world_message();
  auto const message_span = host_span<char>(message.data(), message.size());

  expect_match("hello", message_span.first(5));
}

TEST(SpanTest, CanTakeLast)
{
  auto message            = create_hello_world_message();
  auto const message_span = host_span<char>(message.data(), message.size());

  expect_match("world", message_span.last(5));
}

TEST(SpanTest, CanTakeSubspanFull)
{
  auto message            = create_hello_world_message();
  auto const message_span = host_span<char>(message.data(), message.size());

  expect_match("hello world", message_span.subspan(0, 11));
}

TEST(SpanTest, CanTakeSubspanPartial)
{
  auto message            = create_hello_world_message();
  auto const message_span = host_span<char>(message.data(), message.size());

  expect_match("lo w", message_span.subspan(3, 4));
}

TEST(SpanTest, CanGetFront)
{
  auto message            = create_hello_world_message();
  auto const message_span = host_span<char>(message.data(), message.size());

  EXPECT_EQ('h', message_span.front());
}

TEST(SpanTest, CanGetBack)
{
  auto message            = create_hello_world_message();
  auto const message_span = host_span<char>(message.data(), message.size());

  EXPECT_EQ('d', message_span.back());
}

TEST(SpanTest, CanGetData)
{
  auto message            = create_hello_world_message();
  auto const message_span = host_span<char>(message.data(), message.size());

  EXPECT_EQ(message.data(), message_span.data());
}

TEST(SpanTest, CanDetermineEmptiness)
{
  auto message            = create_hello_world_message();
  auto const message_span = host_span<char>(message.data(), message.size());
  auto const empty_span   = host_span<char>();

  EXPECT_FALSE(message_span.empty());
  EXPECT_TRUE(empty_span.empty());
}

TEST(SpanTest, CanGetSize)
{
  auto message            = create_hello_world_message();
  auto const message_span = host_span<char>(message.data(), message.size());
  auto const empty_span   = host_span<char>();

  EXPECT_EQ(static_cast<size_t>(11), message_span.size());
  EXPECT_EQ(static_cast<size_t>(0), empty_span.size());
}

TEST(SpanTest, CanGetSizeBytes)
{
  auto doubles            = std::vector<double>({6, 3, 2});
  auto const doubles_span = host_span<double>(doubles.data(), doubles.size());
  auto const empty_span   = host_span<double>();

  EXPECT_EQ(static_cast<size_t>(24), doubles_span.size_bytes());
  EXPECT_EQ(static_cast<size_t>(0), empty_span.size_bytes());
}

TEST(SpanTest, CanCopySpan)
{
  auto message = create_hello_world_message();
  host_span<char> message_span_copy;

  {
    auto const message_span = host_span<char>(message.data(), message.size());

    message_span_copy = message_span;
  }

  EXPECT_EQ(message.data(), message_span_copy.data());
  EXPECT_EQ(message.size(), message_span_copy.size());
}

TEST(SpanTest, CanSubscriptRead)
{
  auto message            = create_hello_world_message();
  auto const message_span = host_span<char>(message.data(), message.size());

  EXPECT_EQ('o', message_span[4]);
}

TEST(SpanTest, CanSubscriptWrite)
{
  auto message            = create_hello_world_message();
  auto const message_span = host_span<char>(message.data(), message.size());

  message_span[4] = 'x';

  EXPECT_EQ('x', message_span[4]);
}

TEST(SpanTest, CanConstructFromHostContainers)
{
  auto std_vector = std::vector<int>(1);
  auto h_vector   = thrust::host_vector<int>(1);

  (void)host_span<int>(std_vector);
  (void)host_span<int>(h_vector);

  auto const std_vector_c = std_vector;
  auto const h_vector_c   = h_vector;

  (void)host_span<int const>(std_vector_c);
  (void)host_span<int const>(h_vector_c);
}

// This test is the only place in libcudf's test suite where using a
// thrust::device_vector (and therefore the CUDA default stream) is acceptable
// since we are explicitly testing conversions from thrust::device_vector.
TEST(SpanTest, CanConstructFromDeviceContainers)
{
  auto d_thrust_vector = thrust::device_vector<int>(1);
  auto d_vector        = rmm::device_vector<int>(1);
  auto d_uvector       = rmm::device_uvector<int>(1, cudf::get_default_stream());

  (void)device_span<int>(d_thrust_vector);
  (void)device_span<int>(d_vector);
  (void)device_span<int>(d_uvector);

  auto const& d_thrust_vector_c = d_thrust_vector;
  auto const& d_vector_c        = d_vector;
  auto const& d_uvector_c       = d_uvector;

  (void)device_span<int const>(d_thrust_vector_c);
  (void)device_span<int const>(d_vector_c);
  (void)device_span<int const>(d_uvector_c);
}

CUDF_KERNEL void simple_device_kernel(device_span<bool> result) { result[0] = true; }

TEST(SpanTest, CanUseDeviceSpan)
{
  auto d_message = cudf::detail::make_zeroed_device_uvector_async<bool>(
    1, cudf::get_default_stream(), rmm::mr::get_current_device_resource());

  auto d_span = device_span<bool>(d_message.data(), d_message.size());

  simple_device_kernel<<<1, 1, 0, cudf::get_default_stream().value()>>>(d_span);

  ASSERT_TRUE(d_message.element(0, cudf::get_default_stream()));
}

class MdSpanTest : public cudf::test::BaseFixture {};

TEST(MdSpanTest, CanDetermineEmptiness)
{
  auto const vector            = hostdevice_2dvector<int>(1, 2, cudf::get_default_stream());
  auto const no_rows_vector    = hostdevice_2dvector<int>(0, 2, cudf::get_default_stream());
  auto const no_columns_vector = hostdevice_2dvector<int>(1, 0, cudf::get_default_stream());

  EXPECT_FALSE(host_2dspan<int const>{vector}.is_empty());
  EXPECT_FALSE(device_2dspan<int const>{vector}.is_empty());
  EXPECT_TRUE(host_2dspan<int const>{no_rows_vector}.is_empty());
  EXPECT_TRUE(device_2dspan<int const>{no_rows_vector}.is_empty());
  EXPECT_TRUE(host_2dspan<int const>{no_columns_vector}.is_empty());
  EXPECT_TRUE(device_2dspan<int const>{no_columns_vector}.is_empty());
}

CUDF_KERNEL void readwrite_kernel(device_2dspan<int> result)
{
  if (result[5][6] == 5) {
    result[5][6] *= 6;
  } else {
    result[5][6] = 5;
  }
}

TEST(MdSpanTest, DeviceReadWrite)
{
  auto vector = hostdevice_2dvector<int>(11, 23, cudf::get_default_stream());

  readwrite_kernel<<<1, 1, 0, cudf::get_default_stream().value()>>>(vector);
  readwrite_kernel<<<1, 1, 0, cudf::get_default_stream().value()>>>(vector);
  vector.device_to_host_sync(cudf::get_default_stream());
  EXPECT_EQ(vector[5][6], 30);
}

TEST(MdSpanTest, HostReadWrite)
{
  auto vector = hostdevice_2dvector<int>(11, 23, cudf::get_default_stream());
  auto span   = host_2dspan<int>{vector};
  span[5][6]  = 5;
  if (span[5][6] == 5) { span[5][6] *= 6; }

  EXPECT_EQ(vector[5][6], 30);
}

TEST(MdSpanTest, CanGetSize)
{
  auto const vector = hostdevice_2dvector<int>(1, 2, cudf::get_default_stream());

  EXPECT_EQ(host_2dspan<int const>{vector}.size(), vector.size());
  EXPECT_EQ(device_2dspan<int const>{vector}.size(), vector.size());
}

TEST(MdSpanTest, CanGetCount)
{
  auto const vector = hostdevice_2dvector<int>(11, 23, cudf::get_default_stream());

  EXPECT_EQ(host_2dspan<int const>{vector}.count(), 11ul * 23);
  EXPECT_EQ(device_2dspan<int const>{vector}.count(), 11ul * 23);
}

auto get_test_hostdevice_vector()
{
  auto v = cudf::detail::hostdevice_vector<char>(0, 11, cudf::get_default_stream());
  for (auto c : create_hello_world_message()) {
    v.push_back(c);
  }

  return v;
}

TEST(HostDeviceSpanTest, CanCreateFullSubspan)
{
  auto message = get_test_hostdevice_vector();
  auto const message_span =
    cudf::detail::hostdevice_span<char>(message.host_ptr(), message.device_ptr(), message.size());

  expect_equivalent(message_span, message.subspan(0, message_span.size()));
}

TEST(HostDeviceSpanTest, CanCreateHostSpan)
{
  auto message            = get_test_hostdevice_vector();
  auto const message_span = host_span<char>(message.host_ptr(), message.size());
  auto const hd_span =
    cudf::detail::hostdevice_span<char>(message.host_ptr(), message.device_ptr(), message.size());

  expect_equivalent(message_span, cudf::host_span<char>(hd_span));
}

TEST(HostDeviceSpanTest, CanTakeSubspanFull)
{
  auto message = get_test_hostdevice_vector();
  auto const message_span =
    cudf::detail::hostdevice_span<char>(message.host_ptr(), message.device_ptr(), message.size());

  expect_match("hello world", message.subspan(0, 11));
  expect_match("hello world", message_span.subspan(0, 11));
}

TEST(HostDeviceSpanTest, CanTakeSubspanPartial)
{
  auto message = get_test_hostdevice_vector();
  auto const message_span =
    cudf::detail::hostdevice_span<char>(message.host_ptr(), message.device_ptr(), message.size());

  expect_match("lo w", message.subspan(3, 4));
  expect_match("lo w", message_span.subspan(3, 4));
}

TEST(HostDeviceSpanTest, CanGetData)
{
  auto message = get_test_hostdevice_vector();
  auto const message_span =
    cudf::detail::hostdevice_span<char>(message.host_ptr(), message.device_ptr(), message.size());

  EXPECT_EQ(message.host_ptr(), message_span.host_ptr());
}

TEST(HostDeviceSpanTest, CanGetSize)
{
  auto message = get_test_hostdevice_vector();
  auto const message_span =
    cudf::detail::hostdevice_span<char>(message.host_ptr(), message.device_ptr(), message.size());
  auto const empty_span = cudf::detail::hostdevice_span<char>();

  EXPECT_EQ(static_cast<size_t>(11), message_span.size());
  EXPECT_EQ(static_cast<size_t>(0), empty_span.size());
}

TEST(HostDeviceSpanTest, CanGetSizeBytes)
{
  auto doubles     = std::vector<double>({6, 3, 2});
  auto doubles_hdv = cudf::detail::hostdevice_vector<double>(0, 3, cudf::get_default_stream());
  for (auto d : doubles) {
    doubles_hdv.push_back(d);
  }
  auto const doubles_span = cudf::detail::hostdevice_span<double>(doubles_hdv);
  auto const empty_span   = cudf::detail::hostdevice_span<double>();

  EXPECT_EQ(static_cast<size_t>(24), doubles_span.size_bytes());
  EXPECT_EQ(static_cast<size_t>(0), empty_span.size_bytes());
}

TEST(HostDeviceSpanTest, CanCopySpan)
{
  auto message = get_test_hostdevice_vector();
  cudf::detail::hostdevice_span<char> message_span_copy;

  {
    auto const message_span =
      cudf::detail::hostdevice_span<char>(message.host_ptr(), message.device_ptr(), message.size());

    message_span_copy = message_span;
  }

  EXPECT_EQ(message.host_ptr(), message_span_copy.host_ptr());
  EXPECT_EQ(message.device_ptr(), message_span_copy.device_ptr());
  EXPECT_EQ(message.size(), message_span_copy.size());
}

TEST(HostDeviceSpanTest, CanSendToDevice)
{
  auto message = get_test_hostdevice_vector();

  message.host_to_device_sync(cudf::get_default_stream());

  char d_message[12];
  hipMemcpy(d_message, message.device_ptr(), 11, hipMemcpyDefault);
  d_message[11] = '\0';

  EXPECT_EQ(11, strlen(d_message));
  EXPECT_EQ(std::string(d_message), hello_world_message);
}

CUDF_KERNEL void simple_device_char_kernel(device_span<char> result)
{
  char const* str = "world hello";
  for (int offset = 0; offset < result.size(); ++offset) {
    result.data()[offset] = str[offset];
  }
}

TEST(HostDeviceSpanTest, CanGetFromDevice)
{
  auto message = get_test_hostdevice_vector();
  message.host_to_device_sync(cudf::get_default_stream());
  simple_device_char_kernel<<<1, 1, 0, cudf::get_default_stream()>>>(message);

  message.device_to_host_sync(cudf::get_default_stream());
  expect_match("world hello", cudf::detail::hostdevice_span<char>(message));
}

CUDF_TEST_PROGRAM_MAIN()
