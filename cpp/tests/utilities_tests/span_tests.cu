#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2020, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <cudf/utilities/span.hpp>

#include <cudf_test/base_fixture.hpp>
#include <cudf_test/cudf_gtest.hpp>
#include <cudf_test/type_lists.hpp>

#include <rmm/thrust_rmm_allocator.h>
#include <rmm/device_buffer.hpp>

#include <cstddef>
#include <cstring>
#include <string>

using cudf::detail::device_span;
using cudf::detail::host_span;

template <typename T>
void expect_equivolent(host_span<T> a, host_span<T> b)
{
  EXPECT_EQ(a.size(), b.size());
  EXPECT_EQ(a.data(), b.data());
}

template <typename Iterator1, typename T>
void expect_match(Iterator1 expected, size_t expected_size, host_span<T> input)
{
  EXPECT_EQ(expected_size, input.size());
  for (size_t i = 0; i < expected_size; i++) { EXPECT_EQ(*(expected + i), *(input.begin() + i)); }
}

template <typename T>
void expect_match(std::string expected, host_span<T> input)
{
  return expect_match(expected.begin(), expected.size(), input);
}

std::string const hello_wold_message = "hello world";
std::vector<char> create_hello_world_message()
{
  return std::vector<char>(hello_wold_message.begin(), hello_wold_message.end());
}

class SpanTest : public cudf::test::BaseFixture {
};

TEST(SpanTest, CanCreateFullSubspan)
{
  auto message            = create_hello_world_message();
  auto const message_span = host_span<char>(message.data(), message.size());

  expect_equivolent(message_span, message_span.subspan(0, message_span.size()));
}

TEST(SpanTest, CanTakeFirst)
{
  auto message            = create_hello_world_message();
  auto const message_span = host_span<char>(message.data(), message.size());

  expect_match("hello", message_span.first(5));
}

TEST(SpanTest, CanTakeLast)
{
  auto message            = create_hello_world_message();
  auto const message_span = host_span<char>(message.data(), message.size());

  expect_match("world", message_span.last(5));
}

TEST(SpanTest, CanTakeSubspanFull)
{
  auto message            = create_hello_world_message();
  auto const message_span = host_span<char>(message.data(), message.size());

  expect_match("hello world", message_span.subspan(0, 11));
}

TEST(SpanTest, CanTakeSubspanPartial)
{
  auto message            = create_hello_world_message();
  auto const message_span = host_span<char>(message.data(), message.size());

  expect_match("lo w", message_span.subspan(3, 4));
}

TEST(SpanTest, CanGetFront)
{
  auto message            = create_hello_world_message();
  auto const message_span = host_span<char>(message.data(), message.size());

  EXPECT_EQ('h', message_span.front());
}

TEST(SpanTest, CanGetBack)
{
  auto message            = create_hello_world_message();
  auto const message_span = host_span<char>(message.data(), message.size());

  EXPECT_EQ('d', message_span.back());
}

TEST(SpanTest, CanGetData)
{
  auto message            = create_hello_world_message();
  auto const message_span = host_span<char>(message.data(), message.size());

  EXPECT_EQ(message.data(), message_span.data());
}

TEST(SpanTest, CanDetermineEmptiness)
{
  auto message            = create_hello_world_message();
  auto const message_span = host_span<char>(message.data(), message.size());
  auto const empty_span   = host_span<char>();

  EXPECT_FALSE(message_span.empty());
  EXPECT_TRUE(empty_span.empty());
}

TEST(SpanTest, CanGetSize)
{
  auto message            = create_hello_world_message();
  auto const message_span = host_span<char>(message.data(), message.size());
  auto const empty_span   = host_span<char>();

  EXPECT_EQ(static_cast<size_t>(11), message_span.size());
  EXPECT_EQ(static_cast<size_t>(0), empty_span.size());
}

TEST(SpanTest, CanGetSizeBytes)
{
  auto doubles            = std::vector<double>({6, 3, 2});
  auto const doubles_span = host_span<double>(doubles.data(), doubles.size());
  auto const empty_span   = host_span<double>();

  EXPECT_EQ(static_cast<size_t>(24), doubles_span.size_bytes());
  EXPECT_EQ(static_cast<size_t>(0), empty_span.size_bytes());
}

TEST(SpanTest, CanCopySpan)
{
  auto message = create_hello_world_message();
  host_span<char> message_span_copy;

  {
    auto const message_span = host_span<char>(message.data(), message.size());

    message_span_copy = message_span;
  }

  EXPECT_EQ(message.data(), message_span_copy.data());
  EXPECT_EQ(message.size(), message_span_copy.size());
}

TEST(SpanTest, CanSubscriptRead)
{
  auto message            = create_hello_world_message();
  auto const message_span = host_span<char>(message.data(), message.size());

  EXPECT_EQ('o', message_span[4]);
}

TEST(SpanTest, CanSubscriptWrite)
{
  auto message            = create_hello_world_message();
  auto const message_span = host_span<char>(message.data(), message.size());

  message_span[4] = 'x';

  EXPECT_EQ('x', message_span[4]);
}

TEST(SpanTest, CanConstructFromHostContainers)
{
  auto std_vector = std::vector<int>(1);
  auto h_vector   = thrust::host_vector<int>(1);

  (void)host_span<int>(std_vector);
  (void)host_span<int>(h_vector);

  auto const std_vector_c = std_vector;
  auto const h_vector_c   = h_vector;

  (void)host_span<int const>(std_vector_c);
  (void)host_span<int const>(h_vector_c);
}

TEST(SpanTest, CanConstructFromDeviceContainers)
{
  auto d_thrust_vector = thrust::device_vector<int>(1);
  auto d_vector        = rmm::device_vector<int>(1);
  auto d_uvector       = rmm::device_uvector<int>(1, 0);

  (void)device_span<int>(d_thrust_vector);
  (void)device_span<int>(d_vector);
  (void)device_span<int>(d_uvector);

  auto const& d_thrust_vector_c = d_thrust_vector;
  auto const& d_vector_c        = d_vector;
  auto const& d_uvector_c       = d_uvector;

  (void)device_span<int const>(d_thrust_vector_c);
  (void)device_span<int const>(d_vector_c);
  (void)device_span<int const>(d_uvector_c);
}

__global__ void simple_device_kernel(device_span<bool> result) { result[0] = true; }

TEST(SpanTest, CanUseDeviceSpan)
{
  rmm::device_vector<bool> d_message = std::vector<bool>({false});

  auto d_span = device_span<bool>(d_message.data().get(), d_message.size());

  simple_device_kernel<<<1, 1>>>(d_span);

  hipDeviceSynchronize();

  thrust::host_vector<bool> h_message = d_message;

  ASSERT_TRUE(h_message[0]);
}

CUDF_TEST_PROGRAM_MAIN()
