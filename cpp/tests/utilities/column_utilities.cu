#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019-2020, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <cudf/column/column_view.hpp>
#include <cudf/detail/copy.hpp>
#include <cudf/detail/get_value.cuh>
#include <cudf/dictionary/dictionary_column_view.hpp>
#include <cudf/lists/lists_column_view.hpp>
#include <cudf/strings/convert/convert_datetime.hpp>
#include <cudf/structs/struct_view.hpp>
#include <cudf/structs/structs_column_view.hpp>
#include <cudf/table/row_operators.cuh>
#include <cudf/table/table_device_view.cuh>
#include <cudf/utilities/bit.hpp>
#include "cudf/utilities/type_dispatcher.hpp"

#include <cudf_test/column_utilities.hpp>
#include <cudf_test/column_wrapper.hpp>
#include <cudf_test/cudf_gtest.hpp>
#include <cudf_test/detail/column_utilities.hpp>

#include <jit/type.h>

#include <thrust/equal.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/logical.h>

#include <numeric>
#include <sstream>

namespace cudf {
namespace test {

namespace {

template <bool check_exact_equality>
struct column_property_comparator {
  bool types_equivalent(cudf::data_type const& lhs, cudf::data_type const& rhs)
  {
    return is_fixed_point(lhs) ? lhs.id() == rhs.id() : lhs == rhs;
  }

  void compare_common(cudf::column_view const& lhs, cudf::column_view const& rhs)
  {
    if (check_exact_equality) {
      EXPECT_EQ(lhs.type(), rhs.type());
    } else {
      EXPECT_TRUE(types_equivalent(lhs.type(), rhs.type()));
    }

    EXPECT_EQ(lhs.size(), rhs.size());

    if (lhs.size() > 0 && check_exact_equality) { EXPECT_EQ(lhs.nullable(), rhs.nullable()); }

    // equivalent, but not exactly equal columns can have a different number of children if their
    // sizes are both 0. Specifically, empty string columns may or may not have children.
    if (check_exact_equality || lhs.size() > 0) {
      EXPECT_EQ(lhs.num_children(), rhs.num_children());
    }
  }

  template <typename T, std::enable_if_t<!std::is_same<T, cudf::list_view>::value>* = nullptr>
  void operator()(cudf::column_view const& lhs, cudf::column_view const& rhs)
  {
    compare_common(lhs, rhs);
  }

  template <typename T, std::enable_if_t<std::is_same<T, cudf::list_view>::value>* = nullptr>
  void operator()(cudf::column_view const& lhs, cudf::column_view const& rhs)
  {
    compare_common(lhs, rhs);

    cudf::lists_column_view lhs_l(lhs);
    cudf::lists_column_view rhs_l(rhs);

    // recurse
    cudf::type_dispatcher(lhs_l.child().type(),
                          column_property_comparator<check_exact_equality>{},
                          lhs_l.get_sliced_child(0),
                          rhs_l.get_sliced_child(0));
  }
};

class corresponding_rows_unequal {
 public:
  corresponding_rows_unequal(table_device_view d_lhs, table_device_view d_rhs) : comp(d_lhs, d_rhs)
  {
  }

  cudf::row_equality_comparator<true> comp;

  __device__ bool operator()(size_type index) { return !comp(index, index); }
};

class corresponding_rows_not_equivalent {
  table_device_view d_lhs;
  table_device_view d_rhs;

 public:
  corresponding_rows_not_equivalent(table_device_view d_lhs, table_device_view d_rhs)
    : d_lhs(d_lhs), d_rhs(d_rhs), comp(d_lhs, d_rhs)
  {
    CUDF_EXPECTS(d_lhs.num_columns() == 1 and d_rhs.num_columns() == 1,
                 "Unsupported number of columns");
  }

  struct typed_element_not_equivalent {
    template <typename T>
    __device__ std::enable_if_t<std::is_floating_point<T>::value, bool> operator()(
      column_device_view const& lhs, column_device_view const& rhs, size_type index)
    {
      if (lhs.is_valid(index) and rhs.is_valid(index)) {
        int ulp = 4;  // value taken from google test
        T x     = lhs.element<T>(index);
        T y     = rhs.element<T>(index);
        return std::abs(x - y) > std::numeric_limits<T>::epsilon() * std::abs(x + y) * ulp &&
               std::abs(x - y) >= std::numeric_limits<T>::min();
      } else {
        // if either is null, then the inequality was checked already
        return true;
      }
    }

    template <typename T, typename... Args>
    __device__ std::enable_if_t<not std::is_floating_point<T>::value, bool> operator()(Args... args)
    {
      // Non-floating point inequality is checked already
      return true;
    }
  };

  cudf::row_equality_comparator<true> comp;

  __device__ bool operator()(size_type index)
  {
    if (not comp(index, index)) {
      auto lhs_col = this->d_lhs.column(0);
      auto rhs_col = this->d_rhs.column(0);
      return type_dispatcher(
        lhs_col.type(), typed_element_not_equivalent{}, lhs_col, rhs_col, index);
    }
    return false;
  }
};

std::string differences_message(thrust::device_vector<int> const& differences,
                                column_view const& lhs,
                                column_view const& rhs,
                                bool all_differences,
                                int depth)
{
  CUDF_EXPECTS(not differences.empty(), "Shouldn't enter this function if `differences` is empty");

  std::string const depth_str = depth > 0 ? "depth " + std::to_string(depth) + '\n' : "";

  if (all_differences) {
    std::ostringstream buffer;
    buffer << depth_str << "differences:" << std::endl;

    auto source_table = cudf::table_view({lhs, rhs});
    auto diff_column  = fixed_width_column_wrapper<int32_t>(differences.begin(), differences.end());
    auto diff_table   = cudf::gather(source_table, diff_column);

    //  Need to pull back the differences
    auto const h_left_strings  = to_strings(diff_table->get_column(0));
    auto const h_right_strings = to_strings(diff_table->get_column(1));

    for (size_t i = 0; i < differences.size(); ++i)
      buffer << depth_str << "lhs[" << differences[i] << "] = " << h_left_strings[i] << ", rhs["
             << differences[i] << "] = " << h_right_strings[i] << std::endl;

    return buffer.str();
  } else {
    int index = differences[0];  // only stringify first difference

    auto diff_lhs = cudf::detail::slice(lhs, index, index + 1);
    auto diff_rhs = cudf::detail::slice(rhs, index, index + 1);

    return depth_str + "first difference: " + "lhs[" + std::to_string(index) +
           "] = " + to_string(diff_lhs, "") + ", rhs[" + std::to_string(index) +
           "] = " + to_string(diff_rhs, "");
  }
}

// non-nested column types
template <typename T, bool check_exact_equality>
struct column_comparator_impl {
  void operator()(column_view const& lhs,
                  column_view const& rhs,
                  bool print_all_differences,
                  int depth)
  {
    auto d_lhs = cudf::table_device_view::create(table_view{{lhs}});
    auto d_rhs = cudf::table_device_view::create(table_view{{rhs}});

    using ComparatorType = std::conditional_t<check_exact_equality,
                                              corresponding_rows_unequal,
                                              corresponding_rows_not_equivalent>;

    auto differences = thrust::device_vector<int>(lhs.size());  // worst case: everything different
    auto diff_iter   = thrust::copy_if(thrust::device,
                                     thrust::make_counting_iterator(0),
                                     thrust::make_counting_iterator(lhs.size()),
                                     differences.begin(),
                                     ComparatorType(*d_lhs, *d_rhs));

    differences.resize(thrust::distance(differences.begin(), diff_iter));  // shrink back down

    if (not differences.empty())
      GTEST_FAIL() << differences_message(differences, lhs, rhs, print_all_differences, depth);
  }
};

// forward declaration for nested-type recursion.
template <bool check_exact_equality>
struct column_comparator;

// specialization for list columns
template <bool check_exact_equality>
struct column_comparator_impl<list_view, check_exact_equality> {
  void operator()(column_view const& lhs,
                  column_view const& rhs,
                  bool print_all_differences,
                  int depth)
  {
    lists_column_view lhs_l(lhs);
    lists_column_view rhs_l(rhs);

    CUDF_EXPECTS(lhs_l.size() == rhs_l.size(), "List column size mismatch");
    if (lhs_l.is_empty()) { return; }

    // worst case - everything is different
    thrust::device_vector<int> differences(lhs.size());

    // TODO : determine how equals/equivalency should work for columns with divergent underlying
    // data, but equivalent null masks. Example:
    //
    // List<int32_t>:
    // Length : 3
    // Offsets : 0, 3, 5, 5
    // Nulls: 011
    // Children :
    //   1, 2, 3, 4, 5
    //
    // List<int32_t>:
    // Length : 3
    // Offsets : 0, 3, 5, 7
    // Nulls: 011
    // Children :
    //   1, 2, 3, 4, 5, 7, 8
    //
    // These two columns are seemingly equivalent, since their top level rows are the same, with
    // just the last element being null. However, pyArrow will say these are -not- equal and
    // does not appear to have an equivalent() check.  So the question is : should we be handling
    // this case when someone calls expect_columns_equivalent()?

    // compare offsets, taking slicing into account

    // left side
    size_type lhs_shift = cudf::detail::get_value<size_type>(lhs_l.offsets(), lhs_l.offset(), 0);
    auto lhs_offsets    = thrust::make_transform_iterator(
      lhs_l.offsets().begin<size_type>() + lhs_l.offset(),
      [lhs_shift] __device__(size_type offset) { return offset - lhs_shift; });
    auto lhs_valids = thrust::make_transform_iterator(
      thrust::make_counting_iterator(0),
      [mask = lhs_l.null_mask(), offset = lhs_l.offset()] __device__(size_type index) {
        return mask == nullptr ? true : cudf::bit_is_set(mask, index + offset);
      });

    // right side
    size_type rhs_shift = cudf::detail::get_value<size_type>(rhs_l.offsets(), rhs_l.offset(), 0);
    auto rhs_offsets    = thrust::make_transform_iterator(
      rhs_l.offsets().begin<size_type>() + rhs_l.offset(),
      [rhs_shift] __device__(size_type offset) { return offset - rhs_shift; });
    auto rhs_valids = thrust::make_transform_iterator(
      thrust::make_counting_iterator(0),
      [mask = rhs_l.null_mask(), offset = rhs_l.offset()] __device__(size_type index) {
        return mask == nullptr ? true : cudf::bit_is_set(mask, index + offset);
      });

    auto diff_iter = thrust::copy_if(
      thrust::device,
      thrust::make_counting_iterator(0),
      thrust::make_counting_iterator(lhs_l.size() + 1),
      differences.begin(),
      [lhs_offsets, rhs_offsets, lhs_valids, rhs_valids, num_rows = lhs_l.size()] __device__(
        size_type index) {
        // last offset has no validity associated with it
        if (index < num_rows - 1) {
          if (lhs_valids[index] != rhs_valids[index]) { return true; }
          // if validity matches -and- is false, we can ignore the actual values. this
          // is technically not checking "equal()", but it's how the non-list code path handles it
          if (!lhs_valids[index]) { return false; }
        }
        return lhs_offsets[index] == rhs_offsets[index] ? false : true;
      });

    differences.resize(thrust::distance(differences.begin(), diff_iter));  // shrink back down

    if (not differences.empty())
      GTEST_FAIL() << differences_message(differences, lhs, rhs, print_all_differences, depth);

    // recurse
    auto lhs_child = lhs_l.get_sliced_child(0);
    auto rhs_child = rhs_l.get_sliced_child(0);
    cudf::type_dispatcher(lhs_child.type(),
                          column_comparator<check_exact_equality>{},
                          lhs_child,
                          rhs_child,
                          print_all_differences,
                          depth + 1);
  }
};

template <bool check_exact_equality>
struct column_comparator_impl<struct_view, check_exact_equality> {
  void operator()(column_view const& lhs,
                  column_view const& rhs,
                  bool print_all_differences,
                  int depth)
  {
    structs_column_view l_scv(lhs);
    structs_column_view r_scv(rhs);

    std::for_each(thrust::make_counting_iterator(0),
                  thrust::make_counting_iterator(0) + lhs.num_children(),
                  [&](auto i) {
                    column_view lhs_child = l_scv.get_sliced_child(i);
                    column_view rhs_child = r_scv.get_sliced_child(i);
                    cudf::type_dispatcher(lhs_child.type(),
                                          column_comparator<check_exact_equality>{},
                                          lhs_child,
                                          rhs_child,
                                          print_all_differences,
                                          depth + 1);
                  });
  }
};

template <bool check_exact_equality>
struct column_comparator {
  template <typename T>
  void operator()(column_view const& lhs,
                  column_view const& rhs,
                  bool print_all_differences,
                  int depth = 0)
  {
    // compare properties
    cudf::type_dispatcher(lhs.type(), column_property_comparator<check_exact_equality>{}, lhs, rhs);

    // compare values
    column_comparator_impl<T, check_exact_equality> comparator{};
    comparator(lhs, rhs, print_all_differences, depth);
  }
};

}  // namespace

/**
 * @copydoc cudf::test::expect_column_properties_equal
 *
 */
void expect_column_properties_equal(column_view const& lhs, column_view const& rhs)
{
  cudf::type_dispatcher(lhs.type(), column_property_comparator<true>{}, lhs, rhs);
}

/**
 * @copydoc cudf::test::expect_column_properties_equivalent
 *
 */
void expect_column_properties_equivalent(column_view const& lhs, column_view const& rhs)
{
  cudf::type_dispatcher(lhs.type(), column_property_comparator<false>{}, lhs, rhs);
}

/**
 * @copydoc cudf::test::expect_columns_equal
 *
 */
void expect_columns_equal(cudf::column_view const& lhs,
                          cudf::column_view const& rhs,
                          bool print_all_differences)
{
  cudf::type_dispatcher(lhs.type(), column_comparator<true>{}, lhs, rhs, print_all_differences);
}

/**
 * @copydoc cudf::test::expect_columns_equivalent
 *
 */
void expect_columns_equivalent(cudf::column_view const& lhs,
                               cudf::column_view const& rhs,
                               bool print_all_differences)
{
  cudf::type_dispatcher(lhs.type(), column_comparator<false>{}, lhs, rhs, print_all_differences);
}

/**
 * @copydoc cudf::test::expect_equal_buffers
 *
 */
void expect_equal_buffers(void const* lhs, void const* rhs, std::size_t size_bytes)
{
  if (size_bytes > 0) {
    EXPECT_NE(nullptr, lhs);
    EXPECT_NE(nullptr, rhs);
  }
  auto typed_lhs = static_cast<char const*>(lhs);
  auto typed_rhs = static_cast<char const*>(rhs);
  EXPECT_TRUE(thrust::equal(thrust::device, typed_lhs, typed_lhs + size_bytes, typed_rhs));
}

/**
 * @copydoc cudf::test::bitmask_to_host
 *
 */
std::vector<bitmask_type> bitmask_to_host(cudf::column_view const& c)
{
  if (c.nullable()) {
    auto num_bitmasks = bitmask_allocation_size_bytes(c.size()) / sizeof(bitmask_type);
    std::vector<bitmask_type> host_bitmask(num_bitmasks);
    if (c.offset() == 0) {
      CUDA_TRY(hipMemcpy(host_bitmask.data(),
                          c.null_mask(),
                          num_bitmasks * sizeof(bitmask_type),
                          hipMemcpyDeviceToHost));
    } else {
      auto mask = copy_bitmask(c.null_mask(), c.offset(), c.offset() + c.size());
      CUDA_TRY(hipMemcpy(host_bitmask.data(),
                          mask.data(),
                          num_bitmasks * sizeof(bitmask_type),
                          hipMemcpyDeviceToHost));
    }

    return host_bitmask;
  } else {
    return std::vector<bitmask_type>{};
  }
}

namespace {

template <typename T, typename std::enable_if_t<std::is_integral<T>::value>* = nullptr>
static auto numeric_to_string_precise(T value)
{
  return std::to_string(value);
}

template <typename T, typename std::enable_if_t<std::is_floating_point<T>::value>* = nullptr>
static auto numeric_to_string_precise(T value)
{
  std::ostringstream o;
  o << std::setprecision(std::numeric_limits<T>::max_digits10) << value;
  return o.str();
}

static auto duration_suffix(cudf::duration_D) { return " days"; }

static auto duration_suffix(cudf::duration_s) { return " seconds"; }

static auto duration_suffix(cudf::duration_ms) { return " milliseconds"; }

static auto duration_suffix(cudf::duration_us) { return " microseconds"; }

static auto duration_suffix(cudf::duration_ns) { return " nanoseconds"; }

std::string get_nested_type_str(cudf::column_view const& view)
{
  if (view.type().id() == cudf::type_id::LIST) {
    lists_column_view lcv(view);
    return cudf::jit::get_type_name(view.type()) + "<" + (get_nested_type_str(lcv.child())) + ">";
  }

  if (view.type().id() == cudf::type_id::STRUCT) {
    std::ostringstream out;

    out << cudf::jit::get_type_name(view.type()) + "<";
    std::transform(view.child_begin(),
                   view.child_end(),
                   std::ostream_iterator<std::string>(out, ","),
                   [&out](auto const col) { return get_nested_type_str(col); });
    out << ">";
    return out.str();
  }

  return cudf::jit::get_type_name(view.type());
}

template <typename NestedColumnView>
std::string nested_offsets_to_string(NestedColumnView const& c, std::string const& delimiter = ", ")
{
  column_view offsets = (c.parent()).child(NestedColumnView::offsets_column_index);
  CUDF_EXPECTS(offsets.type().id() == type_id::INT32,
               "Column does not appear to be an offsets column");
  CUDF_EXPECTS(offsets.offset() == 0, "Offsets column has an internal offset!");
  size_type output_size = c.size() + 1;

  // the first offset value to normalize everything against
  size_type first = cudf::detail::get_value<size_type>(offsets, c.offset(), 0);
  rmm::device_vector<size_type> shifted_offsets(output_size);

  // normalize the offset values for the column offset
  size_type const* d_offsets = offsets.head<size_type>() + c.offset();
  thrust::transform(
    rmm::exec_policy(0)->on(0),
    d_offsets,
    d_offsets + output_size,
    shifted_offsets.begin(),
    [first] __device__(int32_t offset) { return static_cast<size_type>(offset - first); });

  thrust::host_vector<size_type> h_shifted_offsets(shifted_offsets);
  std::ostringstream buffer;
  for (size_t idx = 0; idx < h_shifted_offsets.size(); idx++) {
    buffer << h_shifted_offsets[idx];
    if (idx < h_shifted_offsets.size() - 1) { buffer << delimiter; }
  }
  return buffer.str();
}

struct column_view_printer {
  template <typename Element, typename std::enable_if_t<is_numeric<Element>()>* = nullptr>
  void operator()(cudf::column_view const& col,
                  std::vector<std::string>& out,
                  std::string const& indent)
  {
    auto h_data = cudf::test::to_host<Element>(col);

    out.resize(col.size());

    if (col.nullable()) {
      std::transform(thrust::make_counting_iterator(size_type{0}),
                     thrust::make_counting_iterator(col.size()),
                     out.begin(),
                     [&h_data](auto idx) {
                       return bit_is_set(h_data.second.data(), idx)
                                ? numeric_to_string_precise(h_data.first[idx])
                                : std::string("NULL");
                     });

    } else {
      std::transform(h_data.first.begin(), h_data.first.end(), out.begin(), [](Element el) {
        return numeric_to_string_precise(el);
      });
    }
  }

  template <typename Element, typename std::enable_if_t<is_timestamp<Element>()>* = nullptr>
  void operator()(cudf::column_view const& col,
                  std::vector<std::string>& out,
                  std::string const& indent)
  {
    //
    //  For timestamps, convert timestamp column to column of strings, then
    //  call string version
    //
    auto col_as_strings = cudf::strings::from_timestamps(col);
    if (col_as_strings->size() == 0) { return; }

    this->template operator()<cudf::string_view>(*col_as_strings, out, indent);
  }

  template <typename Element, typename std::enable_if_t<cudf::is_fixed_point<Element>()>* = nullptr>
  void operator()(cudf::column_view const& col,
                  std::vector<std::string>& out,
                  std::string const& indent)
  {
    auto const h_data = cudf::test::to_host<Element>(col);
    std::transform(std::cbegin(h_data.first),
                   std::cend(h_data.first),
                   std::back_inserter(out),
                   [](auto const& fp) { return std::to_string(static_cast<double>(fp)); });
  }

  template <typename Element,
            typename std::enable_if_t<std::is_same<Element, cudf::string_view>::value>* = nullptr>
  void operator()(cudf::column_view const& col,
                  std::vector<std::string>& out,
                  std::string const& indent)
  {
    //
    //  Implementation for strings, call special to_host variant
    //
    if (col.is_empty()) return;
    auto h_data = cudf::test::to_host<std::string>(col);

    out.resize(col.size());
    std::transform(thrust::make_counting_iterator(size_type{0}),
                   thrust::make_counting_iterator(col.size()),
                   out.begin(),
                   [&h_data](auto idx) {
                     return h_data.second.empty() || bit_is_set(h_data.second.data(), idx)
                              ? h_data.first[idx]
                              : std::string("NULL");
                   });
  }

  template <typename Element,
            typename std::enable_if_t<std::is_same<Element, cudf::dictionary32>::value>* = nullptr>
  void operator()(cudf::column_view const& col,
                  std::vector<std::string>& out,
                  std::string const& indent)
  {
    cudf::dictionary_column_view dictionary(col);
    if (col.is_empty()) return;
    std::vector<std::string> keys    = to_strings(dictionary.keys());
    std::vector<std::string> indices = to_strings({dictionary.indices().type(),
                                                   dictionary.size(),
                                                   dictionary.indices().head(),
                                                   dictionary.null_mask(),
                                                   dictionary.null_count(),
                                                   dictionary.offset()});
    out.insert(out.end(), keys.begin(), keys.end());
    if (!indices.empty()) {
      std::string first = "\x08 : " + indices.front();  // use : as delimiter
      out.push_back(first);                             // between keys and indices
      out.insert(out.end(), indices.begin() + 1, indices.end());
    }
  }

  // Print the tick counts with the units
  template <typename Element, typename std::enable_if_t<is_duration<Element>()>* = nullptr>
  void operator()(cudf::column_view const& col,
                  std::vector<std::string>& out,
                  std::string const& indent)
  {
    auto h_data = cudf::test::to_host<Element>(col);

    out.resize(col.size());

    if (col.nullable()) {
      std::transform(thrust::make_counting_iterator(size_type{0}),
                     thrust::make_counting_iterator(col.size()),
                     out.begin(),
                     [&h_data](auto idx) {
                       return bit_is_set(h_data.second.data(), idx)
                                ? numeric_to_string_precise(h_data.first[idx].count()) +
                                    duration_suffix(h_data.first[idx])
                                : std::string("NULL");
                     });

    } else {
      std::transform(h_data.first.begin(), h_data.first.end(), out.begin(), [](Element el) {
        return numeric_to_string_precise(el.count()) + duration_suffix(el);
      });
    }
  }

  template <typename Element,
            typename std::enable_if_t<std::is_same<Element, cudf::list_view>::value>* = nullptr>
  void operator()(cudf::column_view const& col,
                  std::vector<std::string>& out,
                  std::string const& indent)
  {
    lists_column_view lcv(col);

    // propage slicing to the child if necessary
    column_view child    = lcv.get_sliced_child(0);
    bool const is_sliced = lcv.offset() > 0 || child.offset() > 0;

    std::string tmp =
      get_nested_type_str(col) + (is_sliced ? "(sliced)" : "") + ":\n" + indent +
      "Length : " + std::to_string(lcv.size()) + "\n" + indent +
      "Offsets : " + (lcv.size() > 0 ? nested_offsets_to_string(lcv) : "") + "\n" +
      (lcv.has_nulls() ? indent + "Null count: " + std::to_string(lcv.null_count()) + "\n" +
                           detail::to_string(bitmask_to_host(col), col.size(), indent) + "\n"
                       : "") +
      indent + "Children :\n" +
      (child.type().id() != type_id::LIST && child.has_nulls()
         ? indent + detail::to_string(bitmask_to_host(child), child.size(), indent) + "\n"
         : "") +
      (detail::to_string(child, ", ", indent + "   ")) + "\n";

    out.push_back(tmp);
  }

  template <typename Element,
            typename std::enable_if_t<std::is_same<Element, cudf::struct_view>::value>* = nullptr>
  void operator()(cudf::column_view const& col,
                  std::vector<std::string>& out,
                  std::string const& indent)
  {
    structs_column_view view{col};

    std::ostringstream out_stream;

    out_stream << get_nested_type_str(col) << ":\n"
               << indent << "Length : " << view.size() << ":\n";
    if (view.has_nulls()) {
      out_stream << indent << "Null count: " << view.null_count() << "\n"
                 << detail::to_string(bitmask_to_host(col), col.size(), indent) << "\n";
    }

    auto iter = thrust::make_counting_iterator(0);
    std::transform(iter,
                   iter + view.num_children(),
                   std::ostream_iterator<std::string>(out_stream, "\n"),
                   [&](size_type index) {
                     return detail::to_string(view.get_sliced_child(index), ", ", indent + "    ");
                   });

    out.push_back(out_stream.str());
  }
};

}  // namespace

namespace detail {

/**
 * @copydoc cudf::test::detail::to_strings
 *
 */
std::vector<std::string> to_strings(cudf::column_view const& col, std::string const& indent)
{
  std::vector<std::string> reply;
  cudf::type_dispatcher(col.type(), column_view_printer{}, col, reply, indent);
  return reply;
}

/**
 * @copydoc cudf::test::detail::to_string(cudf::column_view, std::string, std::string)
 *
 * @param indent Indentation for all output
 */
std::string to_string(cudf::column_view const& col,
                      std::string const& delimiter,
                      std::string const& indent)
{
  std::ostringstream buffer;
  std::vector<std::string> h_data = to_strings(col, indent);

  buffer << indent;
  std::copy(h_data.begin(),
            h_data.end() - (!h_data.empty()),
            std::ostream_iterator<std::string>(buffer, delimiter.c_str()));
  if (!h_data.empty()) buffer << h_data.back();

  return buffer.str();
}

/**
 * @copydoc cudf::test::detail::to_string(std::vector<bitmask_type>, size_type, std::string)
 *
 * @param indent Indentation for all output.  See comment in `to_strings` for
 * a detailed description.
 */
std::string to_string(std::vector<bitmask_type> const& null_mask,
                      size_type null_mask_size,
                      std::string const& indent)
{
  std::ostringstream buffer;
  buffer << indent;
  for (int idx = null_mask_size - 1; idx >= 0; idx--) {
    buffer << (cudf::bit_is_set(null_mask.data(), idx) ? "1" : "0");
  }
  return buffer.str();
}

}  // namespace detail

/**
 * @copydoc cudf::test::to_strings
 *
 */
std::vector<std::string> to_strings(cudf::column_view const& col)
{
  return detail::to_strings(col);
}

/**
 * @copydoc cudf::test::to_string(cudf::column_view, std::string)
 *
 */
std::string to_string(cudf::column_view const& col, std::string const& delimiter)
{
  return detail::to_string(col, delimiter);
}

/**
 * @copydoc cudf::test::to_string(std::vector<bitmask_type>, size_type)
 *
 */
std::string to_string(std::vector<bitmask_type> const& null_mask, size_type null_mask_size)
{
  return detail::to_string(null_mask, null_mask_size);
}

/**
 * @copydoc cudf::test::print
 *
 */
void print(cudf::column_view const& col, std::ostream& os, std::string const& delimiter)
{
  os << to_string(col, delimiter) << std::endl;
}

/**
 * @copydoc cudf::test::validate_host_masks
 *
 */
bool validate_host_masks(std::vector<bitmask_type> const& expected_mask,
                         std::vector<bitmask_type> const& got_mask,
                         size_type number_of_elements)
{
  return std::all_of(thrust::make_counting_iterator(0),
                     thrust::make_counting_iterator(number_of_elements),
                     [&expected_mask, &got_mask](auto index) {
                       return cudf::bit_is_set(expected_mask.data(), index) ==
                              cudf::bit_is_set(got_mask.data(), index);
                     });
}

}  // namespace test
}  // namespace cudf
