#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019-2020, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "column_utilities.hpp"

#include <cudf/column/column_view.hpp>
#include <cudf/detail/copy.hpp>
#include <cudf/dictionary/dictionary_column_view.hpp>
#include <cudf/strings/convert/convert_datetime.hpp>
#include <cudf/table/row_operators.cuh>
#include <cudf/table/table_device_view.cuh>
#include <cudf/utilities/bit.hpp>

#include <tests/utilities/column_wrapper.hpp>
#include <tests/utilities/cudf_gtest.hpp>

#include <thrust/equal.h>
#include <thrust/logical.h>

#include <gmock/gmock.h>
#include <numeric>

namespace cudf {
namespace test {

// Property comparison
template <bool check_exact_equality>
void column_property_comparison(cudf::column_view const& lhs, cudf::column_view const& rhs)
{
  EXPECT_EQ(lhs.type(), rhs.type());
  EXPECT_EQ(lhs.size(), rhs.size());
  if (lhs.size() > 0 and check_exact_equality) { EXPECT_EQ(lhs.nullable(), rhs.nullable()); }
  EXPECT_EQ(lhs.num_children(), rhs.num_children());

  // TODO: compare children properties?
}

void expect_column_properties_equal(column_view const& lhs, column_view const& rhs)
{
  column_property_comparison<true>(lhs, rhs);
}

void expect_column_properties_equivalent(column_view const& lhs, column_view const& rhs)
{
  column_property_comparison<false>(lhs, rhs);
}

class corresponding_rows_unequal {
 public:
  corresponding_rows_unequal(table_device_view d_lhs, table_device_view d_rhs) : comp(d_lhs, d_rhs)
  {
  }

  cudf::experimental::row_equality_comparator<true> comp;

  __device__ bool operator()(size_type index) { return !comp(index, index); }
};

class corresponding_rows_not_equivalent {
  table_device_view d_lhs;
  table_device_view d_rhs;

 public:
  corresponding_rows_not_equivalent(table_device_view d_lhs, table_device_view d_rhs)
    : d_lhs(d_lhs), d_rhs(d_rhs), comp(d_lhs, d_rhs)
  {
    CUDF_EXPECTS(d_lhs.num_columns() == 1 and d_rhs.num_columns() == 1,
                 "Unsupported number of columns");
  }

  struct typed_element_not_equivalent {
    template <typename T>
    __device__ std::enable_if_t<std::is_floating_point<T>::value, bool> operator()(
      column_device_view const& lhs, column_device_view const& rhs, size_type index)
    {
      if (lhs.is_valid(index) and rhs.is_valid(index)) {
        int ulp = 4;  // value taken from google test
        T x     = lhs.element<T>(index);
        T y     = rhs.element<T>(index);
        return std::abs(x - y) > std::numeric_limits<T>::epsilon() * std::abs(x + y) * ulp &&
               std::abs(x - y) >= std::numeric_limits<T>::min();
      } else {
        // if either is null, then the inequality was checked already
        return true;
      }
    }

    template <typename T, typename... Args>
    __device__ std::enable_if_t<not std::is_floating_point<T>::value, bool> operator()(Args... args)
    {
      // Non-floating point inequality is checked already
      return true;
    }
  };

  cudf::experimental::row_equality_comparator<true> comp;

  __device__ bool operator()(size_type index)
  {
    if (not comp(index, index)) {
      auto lhs_col = this->d_lhs.column(0);
      auto rhs_col = this->d_rhs.column(0);
      return experimental::type_dispatcher(
        lhs_col.type(), typed_element_not_equivalent{}, lhs_col, rhs_col, index);
    }
    return false;
  }
};

namespace {

template <bool check_exact_equality>
void column_comparison(cudf::column_view const& lhs,
                       cudf::column_view const& rhs,
                       bool print_all_differences)
{
  column_property_comparison<check_exact_equality>(lhs, rhs);

  using ComparatorType = std::conditional_t<check_exact_equality,
                                            corresponding_rows_unequal,
                                            corresponding_rows_not_equivalent>;

  auto d_lhs = cudf::table_device_view::create(table_view{{lhs}});
  auto d_rhs = cudf::table_device_view::create(table_view{{rhs}});

  thrust::device_vector<int> differences(lhs.size());

  auto diff_iter = thrust::copy_if(thrust::device,
                                   thrust::make_counting_iterator(0),
                                   thrust::make_counting_iterator(lhs.size()),
                                   differences.begin(),
                                   ComparatorType(*d_lhs, *d_rhs));

  differences.resize(thrust::distance(differences.begin(), diff_iter));

  if (diff_iter > differences.begin()) {
    if (print_all_differences) {
      //
      //  If there are differences, display them all
      //
      std::ostringstream buffer;
      buffer << "differences:" << std::endl;

      cudf::table_view source_table({lhs, rhs});

      fixed_width_column_wrapper<int32_t> diff_column(differences.begin(), differences.end());

      std::unique_ptr<cudf::experimental::table> diff_table =
        cudf::experimental::gather(source_table, diff_column);

      //
      //  Need to pull back the differences
      //
      std::vector<std::string> h_left_strings  = to_strings(diff_table->get_column(0));
      std::vector<std::string> h_right_strings = to_strings(diff_table->get_column(1));

      for (size_t i = 0; i < differences.size(); ++i) {
        buffer << "lhs[" << differences[i] << "] = " << h_left_strings[i] << ", rhs["
               << differences[i] << "] = " << h_right_strings[i] << std::endl;
      }

      EXPECT_EQ(differences.size(), size_t{0}) << buffer.str();
    } else {
      //
      //  If there are differences, just display the first one
      //
      int index = differences[0];

      auto diff_lhs = cudf::experimental::detail::slice(lhs, index, index + 1);
      auto diff_rhs = cudf::experimental::detail::slice(rhs, index, index + 1);

      std::vector<std::string> h_left_strings  = to_strings(diff_lhs);
      std::vector<std::string> h_right_strings = to_strings(diff_rhs);

      EXPECT_EQ(differences.size(), size_t{0})
        << "first difference: "
        << "lhs[" << index << "] = " << to_string(diff_lhs, "") << ", rhs[" << index
        << "] = " << to_string(diff_rhs, "");
    }
  }
}

}  // namespace

void expect_columns_equal(cudf::column_view const& lhs,
                          cudf::column_view const& rhs,
                          bool print_all_differences)
{
  column_comparison<true>(lhs, rhs, print_all_differences);
}

void expect_columns_equivalent(cudf::column_view const& lhs,
                               cudf::column_view const& rhs,
                               bool print_all_differences)
{
  column_comparison<false>(lhs, rhs, print_all_differences);
}

// Bitwise equality
void expect_equal_buffers(void const* lhs, void const* rhs, std::size_t size_bytes)
{
  if (size_bytes > 0) {
    EXPECT_NE(nullptr, lhs);
    EXPECT_NE(nullptr, rhs);
  }
  auto typed_lhs = static_cast<char const*>(lhs);
  auto typed_rhs = static_cast<char const*>(rhs);
  EXPECT_TRUE(thrust::equal(thrust::device, typed_lhs, typed_lhs + size_bytes, typed_rhs));
}

// copy column bitmask to host (used by to_host())
std::vector<bitmask_type> bitmask_to_host(cudf::column_view const& c)
{
  if (c.nullable()) {
    auto num_bitmasks = bitmask_allocation_size_bytes(c.size()) / sizeof(bitmask_type);
    std::vector<bitmask_type> host_bitmask(num_bitmasks);
    if (c.offset() == 0) {
      CUDA_TRY(hipMemcpy(host_bitmask.data(),
                          c.null_mask(),
                          num_bitmasks * sizeof(bitmask_type),
                          hipMemcpyDeviceToHost));
    } else {
      auto mask = copy_bitmask(c.null_mask(), c.offset(), c.offset() + c.size());
      CUDA_TRY(hipMemcpy(host_bitmask.data(),
                          mask.data(),
                          num_bitmasks * sizeof(bitmask_type),
                          hipMemcpyDeviceToHost));
    }

    return host_bitmask;
  } else {
    return std::vector<bitmask_type>{};
  }
}

template <typename T, typename std::enable_if_t<std::is_integral<T>::value>* = nullptr>
static auto numeric_to_string_precise(T value)
{
  return std::to_string(value);
}

template <typename T, typename std::enable_if_t<std::is_floating_point<T>::value>* = nullptr>
static auto numeric_to_string_precise(T value)
{
  std::ostringstream o;
  o << std::setprecision(std::numeric_limits<T>::max_digits10) << value;
  return o.str();
}

struct column_view_printer {
  template <typename Element, typename std::enable_if_t<is_numeric<Element>()>* = nullptr>
  void operator()(cudf::column_view const& col, std::vector<std::string>& out)
  {
    auto h_data = cudf::test::to_host<Element>(col);

    out.resize(col.size());

    if (col.nullable()) {
      std::transform(thrust::make_counting_iterator(size_type{0}),
                     thrust::make_counting_iterator(col.size()),
                     out.begin(),
                     [&h_data](auto idx) {
                       return bit_is_set(h_data.second.data(), idx)
                                ? numeric_to_string_precise(h_data.first[idx])
                                : std::string("NULL");
                     });

    } else {
      std::transform(h_data.first.begin(), h_data.first.end(), out.begin(), [](Element el) {
        return numeric_to_string_precise(el);
      });
    }
  }

  template <typename Element, typename std::enable_if_t<is_timestamp<Element>()>* = nullptr>
  void operator()(cudf::column_view const& col, std::vector<std::string>& out)
  {
    //
    //  For timestamps, convert timestamp column to column of strings, then
    //  call string version
    //
    auto col_as_strings = cudf::strings::from_timestamps(col);

    this->template operator()<cudf::string_view>(*col_as_strings, out);
  }

  template <typename Element,
            typename std::enable_if_t<std::is_same<Element, cudf::string_view>::value>* = nullptr>
  void operator()(cudf::column_view const& col, std::vector<std::string>& out)
  {
    //
    //  Implementation for strings, call special to_host variant
    //
    auto h_data = cudf::test::to_host<std::string>(col);

    out.resize(col.size());
    std::transform(thrust::make_counting_iterator(size_type{0}),
                   thrust::make_counting_iterator(col.size()),
                   out.begin(),
                   [&h_data](auto idx) {
                     return h_data.second.empty() || bit_is_set(h_data.second.data(), idx)
                              ? h_data.first[idx]
                              : std::string("NULL");
                   });
  }

  template <typename Element,
            typename std::enable_if_t<std::is_same<Element, cudf::dictionary32>::value>* = nullptr>
  void operator()(cudf::column_view const& col, std::vector<std::string>& out)
  {
    cudf::dictionary_column_view dictionary(col);
    if (col.size() == 0) return;
    std::vector<std::string> keys    = to_strings(dictionary.keys());
    std::vector<std::string> indices = to_strings({cudf::data_type{cudf::INT32},
                                                   dictionary.size(),
                                                   dictionary.indices().head<int32_t>(),
                                                   dictionary.null_mask(),
                                                   dictionary.null_count(),
                                                   dictionary.offset()});
    out.insert(out.end(), keys.begin(), keys.end());
    if (!indices.empty()) {
      std::string first = "\x08 : " + indices.front();  // use : as delimiter
      out.push_back(first);                             // between keys and indices
      out.insert(out.end(), indices.begin() + 1, indices.end());
    }
  }
};

std::vector<std::string> to_strings(cudf::column_view const& col)
{
  std::vector<std::string> reply;
  cudf::experimental::type_dispatcher(col.type(), column_view_printer{}, col, reply);
  return reply;
}

std::string to_string(cudf::column_view const& col, std::string const& delimiter)
{
  std::ostringstream buffer;
  std::vector<std::string> h_data = to_strings(col);

  std::copy(h_data.begin(),
            h_data.end() - (!h_data.empty()),
            std::ostream_iterator<std::string>(buffer, delimiter.c_str()));
  if (!h_data.empty()) buffer << h_data.back();

  return buffer.str();
}

void print(cudf::column_view const& col, std::ostream& os, std::string const& delimiter)
{
  os << to_string(col, delimiter) << std::endl;
}

bool validate_host_masks(std::vector<bitmask_type> const& expected_mask,
                         std::vector<bitmask_type> const& got_mask,
                         size_type number_of_elements)
{
  return std::all_of(thrust::make_counting_iterator(0),
                     thrust::make_counting_iterator(number_of_elements),
                     [&expected_mask, &got_mask](auto index) {
                       return cudf::bit_is_set(expected_mask.data(), index) ==
                              cudf::bit_is_set(got_mask.data(), index);
                     });
}

}  // namespace test
}  // namespace cudf
