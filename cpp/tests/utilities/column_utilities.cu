#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019-2020, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "column_utilities.hpp"
#include "detail/column_utilities.hpp"

#include <cudf/column/column_view.hpp>
#include <cudf/detail/copy.hpp>
#include <cudf/dictionary/dictionary_column_view.hpp>
#include <cudf/lists/lists_column_view.hpp>
#include <cudf/strings/convert/convert_datetime.hpp>
#include <cudf/table/row_operators.cuh>
#include <cudf/table/table_device_view.cuh>
#include <cudf/utilities/bit.hpp>

#include <tests/utilities/column_wrapper.hpp>
#include <tests/utilities/cudf_gtest.hpp>

#include <jit/type.h>

#include <thrust/equal.h>
#include <thrust/logical.h>

#include <numeric>

namespace cudf {
namespace test {

namespace {

template <bool check_exact_equality>
struct column_property_comparator {
  template <typename T>
  void operator()(cudf::column_view const& lhs, cudf::column_view const& rhs)
  {
    EXPECT_EQ(lhs.type(), rhs.type());
    EXPECT_EQ(lhs.size(), rhs.size());
    if (lhs.size() > 0 && check_exact_equality) { EXPECT_EQ(lhs.nullable(), rhs.nullable()); }
    EXPECT_EQ(lhs.num_children(), rhs.num_children());

    // only recurse for true nested types.
    // - strings are an odd case of not being a nested type which do have children. but because
    //   of the way strings handle offsets (sliced/split columns), direct comparison between two
    //   sets of child columns can produce false failures - the sizes may not match.  the truly
    //   correct way to do this would be to implement a specialization for strings (and
    //   dictionaries, lists, etc) that explicitly understand this structure.  but for now, this
    //   seems to be ok.
    if (cudf::is_nested<T>()) {
      for (size_type idx = 0; idx < lhs.num_children(); idx++) {
        cudf::type_dispatcher(lhs.child(idx).type(),
                              column_property_comparator<check_exact_equality>{},
                              lhs.child(idx),
                              rhs.child(idx));
      }
    }
  }
};

class corresponding_rows_unequal {
 public:
  corresponding_rows_unequal(table_device_view d_lhs, table_device_view d_rhs) : comp(d_lhs, d_rhs)
  {
  }

  cudf::row_equality_comparator<true> comp;

  __device__ bool operator()(size_type index) { return !comp(index, index); }
};

class corresponding_rows_not_equivalent {
  table_device_view d_lhs;
  table_device_view d_rhs;

 public:
  corresponding_rows_not_equivalent(table_device_view d_lhs, table_device_view d_rhs)
    : d_lhs(d_lhs), d_rhs(d_rhs), comp(d_lhs, d_rhs)
  {
    CUDF_EXPECTS(d_lhs.num_columns() == 1 and d_rhs.num_columns() == 1,
                 "Unsupported number of columns");
  }

  struct typed_element_not_equivalent {
    template <typename T>
    __device__ std::enable_if_t<std::is_floating_point<T>::value, bool> operator()(
      column_device_view const& lhs, column_device_view const& rhs, size_type index)
    {
      if (lhs.is_valid(index) and rhs.is_valid(index)) {
        int ulp = 4;  // value taken from google test
        T x     = lhs.element<T>(index);
        T y     = rhs.element<T>(index);
        return std::abs(x - y) > std::numeric_limits<T>::epsilon() * std::abs(x + y) * ulp &&
               std::abs(x - y) >= std::numeric_limits<T>::min();
      } else {
        // if either is null, then the inequality was checked already
        return true;
      }
    }

    template <typename T, typename... Args>
    __device__ std::enable_if_t<not std::is_floating_point<T>::value, bool> operator()(Args... args)
    {
      // Non-floating point inequality is checked already
      return true;
    }
  };

  cudf::row_equality_comparator<true> comp;

  __device__ bool operator()(size_type index)
  {
    if (not comp(index, index)) {
      auto lhs_col = this->d_lhs.column(0);
      auto rhs_col = this->d_rhs.column(0);
      return type_dispatcher(
        lhs_col.type(), typed_element_not_equivalent{}, lhs_col, rhs_col, index);
    }
    return false;
  }
};

void print_differences(thrust::device_vector<int> const& differences,
                       column_view const& lhs,
                       column_view const& rhs,
                       bool print_all_differences,
                       int depth)
{
  if (differences.size() <= 0) { return; }

  std::string depth_str = depth > 0 ? "depth " + std::to_string(depth) + std::string("\n") : "";

  if (print_all_differences) {
    //  If there are differences, display them all
    std::ostringstream buffer;
    buffer << depth_str << "differences:" << std::endl;

    cudf::table_view source_table({lhs, rhs});

    fixed_width_column_wrapper<int32_t> diff_column(differences.begin(), differences.end());

    std::unique_ptr<cudf::table> diff_table = cudf::gather(source_table, diff_column);

    //  Need to pull back the differences
    std::vector<std::string> h_left_strings  = to_strings(diff_table->get_column(0));
    std::vector<std::string> h_right_strings = to_strings(diff_table->get_column(1));

    for (size_t i = 0; i < differences.size(); ++i) {
      buffer << depth_str << "lhs[" << differences[i] << "] = " << h_left_strings[i] << ", rhs["
             << differences[i] << "] = " << h_right_strings[i] << std::endl;
    }

    EXPECT_EQ(differences.size(), size_t{0}) << buffer.str();
  } else {
    //  If there are differences, just display the first one
    int index = differences[0];

    auto diff_lhs = cudf::detail::slice(lhs, index, index + 1);
    auto diff_rhs = cudf::detail::slice(rhs, index, index + 1);

    std::vector<std::string> h_left_strings  = to_strings(diff_lhs);
    std::vector<std::string> h_right_strings = to_strings(diff_rhs);

    EXPECT_EQ(differences.size(), size_t{0})
      << depth_str << "first difference: "
      << "lhs[" << index << "] = " << to_string(diff_lhs, "") << ", rhs[" << index
      << "] = " << to_string(diff_rhs, "");
  }
}

// non-nested column types
template <typename T, bool check_exact_equality>
struct column_comparator_impl {
  void operator()(column_view const& lhs,
                  column_view const& rhs,
                  bool print_all_differences,
                  int depth)
  {
    using ComparatorType = std::conditional_t<check_exact_equality,
                                              corresponding_rows_unequal,
                                              corresponding_rows_not_equivalent>;

    auto d_lhs = cudf::table_device_view::create(table_view{{lhs}});
    auto d_rhs = cudf::table_device_view::create(table_view{{rhs}});

    // worst case - everything is different
    thrust::device_vector<int> differences(lhs.size());

    auto diff_iter = thrust::copy_if(thrust::device,
                                     thrust::make_counting_iterator(0),
                                     thrust::make_counting_iterator(lhs.size()),
                                     differences.begin(),
                                     ComparatorType(*d_lhs, *d_rhs));

    // shrink back down
    differences.resize(thrust::distance(differences.begin(), diff_iter));
    print_differences(differences, lhs, rhs, print_all_differences, depth);
  }
};

// forward declaration for nested-type recursion.
template <bool check_exact_equality>
struct column_comparator;

// specialization for list columns
template <bool check_exact_equality>
struct column_comparator_impl<list_view, check_exact_equality> {
  void operator()(column_view const& lhs,
                  column_view const& rhs,
                  bool print_all_differences,
                  int depth)
  {
    lists_column_view lhs_l(lhs);
    lists_column_view rhs_l(rhs);

    // using the row_equality_operator directly on a list column is a bad idea for several
    // reasons:
    // - at the moment, the row_equality_operator doesn't support lists
    //
    // - if it -did-, a "row" in a list column can itself be nested.  so to do a row
    //   comparison involves actually recursing through the hierarchy of data. this recursion
    //   would be happening for each row compared, which is algorithmically terrible.
    //
    // Instead, we can simply walk the hierarchy once, checking each pair of offset columns for
    // equivalency and then finally checking the leaves, which are not nested types.
    cudf::type_dispatcher(lhs_l.offsets().type(),
                          column_comparator<check_exact_equality>{},
                          lhs_l.offsets(),
                          rhs_l.offsets(),
                          print_all_differences,
                          depth);
    cudf::type_dispatcher(lhs_l.child().type(),
                          column_comparator<check_exact_equality>{},
                          lhs_l.child(),
                          rhs_l.child(),
                          print_all_differences,
                          depth + 1);

    // TODO:  to display differences between list columns what we really want to do is
    //        - if there are differences in the leaf values, display those.
    //
    //        otherwise
    //
    //        - determine the first level at which there are list differences (via the offsets),
    //          do a gather on those rows and display them.
  }
};

template <bool check_exact_equality>
struct column_comparator {
  template <typename T>
  void operator()(column_view const& lhs,
                  column_view const& rhs,
                  bool print_all_differences,
                  int depth = 0)
  {
    // compare properties
    cudf::type_dispatcher(lhs.type(), column_property_comparator<check_exact_equality>{}, lhs, rhs);

    // compare values
    column_comparator_impl<T, check_exact_equality> comparator{};
    comparator(lhs, rhs, print_all_differences, depth);
  }
};

}  // namespace

/**
 * @copydoc cudf::test::expect_column_properties_equal
 *
 */
void expect_column_properties_equal(column_view const& lhs, column_view const& rhs)
{
  cudf::type_dispatcher(lhs.type(), column_property_comparator<true>{}, lhs, rhs);
}

/**
 * @copydoc cudf::test::expect_column_properties_equivalent
 *
 */
void expect_column_properties_equivalent(column_view const& lhs, column_view const& rhs)
{
  cudf::type_dispatcher(lhs.type(), column_property_comparator<false>{}, lhs, rhs);
}

/**
 * @copydoc cudf::test::expect_columns_equal
 *
 */
void expect_columns_equal(cudf::column_view const& lhs,
                          cudf::column_view const& rhs,
                          bool print_all_differences)
{
  cudf::type_dispatcher(lhs.type(), column_comparator<true>{}, lhs, rhs, print_all_differences);
}

/**
 * @copydoc cudf::test::expect_columns_equivalent
 *
 */
void expect_columns_equivalent(cudf::column_view const& lhs,
                               cudf::column_view const& rhs,
                               bool print_all_differences)
{
  cudf::type_dispatcher(lhs.type(), column_comparator<false>{}, lhs, rhs, print_all_differences);
}

/**
 * @copydoc cudf::test::expect_equal_buffers
 *
 */
void expect_equal_buffers(void const* lhs, void const* rhs, std::size_t size_bytes)
{
  if (size_bytes > 0) {
    EXPECT_NE(nullptr, lhs);
    EXPECT_NE(nullptr, rhs);
  }
  auto typed_lhs = static_cast<char const*>(lhs);
  auto typed_rhs = static_cast<char const*>(rhs);
  EXPECT_TRUE(thrust::equal(thrust::device, typed_lhs, typed_lhs + size_bytes, typed_rhs));
}

/**
 * @copydoc cudf::test::bitmask_to_host
 *
 */
std::vector<bitmask_type> bitmask_to_host(cudf::column_view const& c)
{
  if (c.nullable()) {
    auto num_bitmasks = bitmask_allocation_size_bytes(c.size()) / sizeof(bitmask_type);
    std::vector<bitmask_type> host_bitmask(num_bitmasks);
    if (c.offset() == 0) {
      CUDA_TRY(hipMemcpy(host_bitmask.data(),
                          c.null_mask(),
                          num_bitmasks * sizeof(bitmask_type),
                          hipMemcpyDeviceToHost));
    } else {
      auto mask = copy_bitmask(c.null_mask(), c.offset(), c.offset() + c.size());
      CUDA_TRY(hipMemcpy(host_bitmask.data(),
                          mask.data(),
                          num_bitmasks * sizeof(bitmask_type),
                          hipMemcpyDeviceToHost));
    }

    return host_bitmask;
  } else {
    return std::vector<bitmask_type>{};
  }
}

namespace {

template <typename T, typename std::enable_if_t<std::is_integral<T>::value>* = nullptr>
static auto numeric_to_string_precise(T value)
{
  return std::to_string(value);
}

template <typename T, typename std::enable_if_t<std::is_floating_point<T>::value>* = nullptr>
static auto numeric_to_string_precise(T value)
{
  std::ostringstream o;
  o << std::setprecision(std::numeric_limits<T>::max_digits10) << value;
  return o.str();
}

std::string get_nested_type_str(cudf::column_view const& view)
{
  if (view.type().id() == cudf::type_id::LIST) {
    lists_column_view lcv(view);
    return cudf::jit::get_type_name(view.type()) + "<" +
           (lcv.size() > 0 ? get_nested_type_str(lcv.child()) : "") + ">";
  }
  return cudf::jit::get_type_name(view.type());
}

struct column_view_printer {
  template <typename Element, typename std::enable_if_t<is_numeric<Element>()>* = nullptr>
  void operator()(cudf::column_view const& col,
                  std::vector<std::string>& out,
                  std::string const& indent)
  {
    auto h_data = cudf::test::to_host<Element>(col);

    out.resize(col.size());

    if (col.nullable()) {
      std::transform(thrust::make_counting_iterator(size_type{0}),
                     thrust::make_counting_iterator(col.size()),
                     out.begin(),
                     [&h_data](auto idx) {
                       return bit_is_set(h_data.second.data(), idx)
                                ? numeric_to_string_precise(h_data.first[idx])
                                : std::string("NULL");
                     });

    } else {
      std::transform(h_data.first.begin(), h_data.first.end(), out.begin(), [](Element el) {
        return numeric_to_string_precise(el);
      });
    }
  }

  template <typename Element, typename std::enable_if_t<is_timestamp<Element>()>* = nullptr>
  void operator()(cudf::column_view const& col,
                  std::vector<std::string>& out,
                  std::string const& indent)
  {
    //
    //  For timestamps, convert timestamp column to column of strings, then
    //  call string version
    //
    auto col_as_strings = cudf::strings::from_timestamps(col);
    if (col_as_strings->size() == 0) { return; }

    this->template operator()<cudf::string_view>(*col_as_strings, out, indent);
  }

  template <typename Element,
            typename std::enable_if_t<std::is_same<Element, cudf::string_view>::value>* = nullptr>
  void operator()(cudf::column_view const& col,
                  std::vector<std::string>& out,
                  std::string const& indent)
  {
    //
    //  Implementation for strings, call special to_host variant
    //
    auto h_data = cudf::test::to_host<std::string>(col);

    out.resize(col.size());
    std::transform(thrust::make_counting_iterator(size_type{0}),
                   thrust::make_counting_iterator(col.size()),
                   out.begin(),
                   [&h_data](auto idx) {
                     return h_data.second.empty() || bit_is_set(h_data.second.data(), idx)
                              ? h_data.first[idx]
                              : std::string("NULL");
                   });
  }

  template <typename Element,
            typename std::enable_if_t<std::is_same<Element, cudf::dictionary32>::value>* = nullptr>
  void operator()(cudf::column_view const& col,
                  std::vector<std::string>& out,
                  std::string const& indent)
  {
    cudf::dictionary_column_view dictionary(col);
    if (col.size() == 0) return;
    std::vector<std::string> keys    = to_strings(dictionary.keys());
    std::vector<std::string> indices = to_strings({cudf::data_type{cudf::type_id::INT32},
                                                   dictionary.size(),
                                                   dictionary.indices().head<int32_t>(),
                                                   dictionary.null_mask(),
                                                   dictionary.null_count(),
                                                   dictionary.offset()});
    out.insert(out.end(), keys.begin(), keys.end());
    if (!indices.empty()) {
      std::string first = "\x08 : " + indices.front();  // use : as delimiter
      out.push_back(first);                             // between keys and indices
      out.insert(out.end(), indices.begin() + 1, indices.end());
    }
  }

  template <typename Element, typename std::enable_if_t<is_duration<Element>()>* = nullptr>
  void operator()(cudf::column_view const& col,
                  std::vector<std::string>& out,
                  std::string const& indent)
  {
    CUDF_FAIL("duration printing not supported yet");
  }

  template <typename Element,
            typename std::enable_if_t<std::is_same<Element, cudf::list_view>::value>* = nullptr>
  void operator()(cudf::column_view const& col,
                  std::vector<std::string>& out,
                  std::string const& indent)
  {
    lists_column_view lcv(col);

    std::string tmp =
      get_nested_type_str(col) + ":\n" + indent + "Length : " + std::to_string(lcv.size()) + "\n" +
      indent + "Offsets : " + (lcv.size() > 0 ? to_string(lcv.offsets(), ", ") : "") + "\n" +
      (lcv.has_nulls() ? indent + "Null count: " + std::to_string(lcv.null_count()) + "\n" +
                           detail::to_string(bitmask_to_host(col), col.size(), indent) + "\n"
                       : "") +
      indent + "Children :\n" +
      (lcv.size() > 0 ? detail::to_string(lcv.child(), ", ", indent + "   ") : "") + "\n";

    out.push_back(tmp);
  }
};

}  // namespace

namespace detail {

/**
 * @copydoc cudf::test::detail::to_strings
 *
 */
std::vector<std::string> to_strings(cudf::column_view const& col, std::string const& indent)
{
  std::vector<std::string> reply;
  cudf::type_dispatcher(col.type(), column_view_printer{}, col, reply, indent);
  return reply;
}

/**
 * @copydoc cudf::test::detail::to_string(cudf::column_view, std::string, std::string)
 *
 * @param indent Indentation for all output
 */
std::string to_string(cudf::column_view const& col,
                      std::string const& delimiter,
                      std::string const& indent)
{
  std::ostringstream buffer;
  std::vector<std::string> h_data = to_strings(col, indent);

  buffer << indent;
  std::copy(h_data.begin(),
            h_data.end() - (!h_data.empty()),
            std::ostream_iterator<std::string>(buffer, delimiter.c_str()));
  if (!h_data.empty()) buffer << h_data.back();

  return buffer.str();
}

/**
 * @copydoc cudf::test::detail::to_string(std::vector<bitmask_type>, size_type, std::string)
 *
 * @param indent Indentation for all output.  See comment in `to_strings` for
 * a detailed description.
 */
std::string to_string(std::vector<bitmask_type> const& null_mask,
                      size_type null_mask_size,
                      std::string const& indent)
{
  std::ostringstream buffer;
  buffer << indent;
  for (int idx = null_mask_size - 1; idx >= 0; idx--) {
    buffer << (cudf::bit_is_set(null_mask.data(), idx) ? "1" : "0");
  }
  return buffer.str();
}

}  // namespace detail

/**
 * @copydoc cudf::test::to_strings
 *
 */
std::vector<std::string> to_strings(cudf::column_view const& col)
{
  return detail::to_strings(col);
}

/**
 * @copydoc cudf::test::to_string(cudf::column_view, std::string)
 *
 */
std::string to_string(cudf::column_view const& col, std::string const& delimiter)
{
  return detail::to_string(col, delimiter);
}

/**
 * @copydoc cudf::test::to_string(std::vector<bitmask_type>, size_type)
 *
 */
std::string to_string(std::vector<bitmask_type> const& null_mask, size_type null_mask_size)
{
  return detail::to_string(null_mask, null_mask_size);
}

/**
 * @copydoc cudf::test::print
 *
 */
void print(cudf::column_view const& col, std::ostream& os, std::string const& delimiter)
{
  os << to_string(col, delimiter) << std::endl;
}

/**
 * @copydoc cudf::test::validate_host_masks
 *
 */
bool validate_host_masks(std::vector<bitmask_type> const& expected_mask,
                         std::vector<bitmask_type> const& got_mask,
                         size_type number_of_elements)
{
  return std::all_of(thrust::make_counting_iterator(0),
                     thrust::make_counting_iterator(number_of_elements),
                     [&expected_mask, &got_mask](auto index) {
                       return cudf::bit_is_set(expected_mask.data(), index) ==
                              cudf::bit_is_set(got_mask.data(), index);
                     });
}

}  // namespace test
}  // namespace cudf
