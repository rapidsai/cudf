/*
 * Copyright (c) 2019-2020, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <tests/utilities/scalar_utilities.hpp>

#include <jit/type.h>
#include <cudf/scalar/scalar.hpp>
#include <cudf/scalar/scalar_factories.hpp>
#include <cudf/utilities/type_dispatcher.hpp>
#include <sstream>
#include <tests/utilities/cudf_gtest.hpp>
#include <type_traits>
#include "gtest/gtest.h"

using cudf::scalar_type_t;

namespace cudf {
namespace test {
namespace {
struct compare_scalar_functor {
  template <typename T>
  void operator()(cudf::scalar const& lhs, cudf::scalar const& rhs)
  {
    auto lhs_t = static_cast<scalar_type_t<T> const&>(lhs);
    auto rhs_t = static_cast<scalar_type_t<T> const&>(rhs);
    EXPECT_EQ(lhs_t.value(), rhs_t.value());
  }
};

template <>
void compare_scalar_functor::operator()<float>(cudf::scalar const& lhs, cudf::scalar const& rhs)
{
  auto lhs_t = static_cast<scalar_type_t<float> const&>(lhs);
  auto rhs_t = static_cast<scalar_type_t<float> const&>(rhs);
  EXPECT_FLOAT_EQ(lhs_t.value(), rhs_t.value());
}

template <>
void compare_scalar_functor::operator()<double>(cudf::scalar const& lhs, cudf::scalar const& rhs)
{
  auto lhs_t = static_cast<scalar_type_t<double> const&>(lhs);
  auto rhs_t = static_cast<scalar_type_t<double> const&>(rhs);
  EXPECT_DOUBLE_EQ(lhs_t.value(), rhs_t.value());
}

template <>
void compare_scalar_functor::operator()<cudf::dictionary32>(cudf::scalar const& lhs,
                                                            cudf::scalar const& rhs)
{
  CUDF_FAIL("Unsupported scalar compare type: dictionary");
}

template <>
void compare_scalar_functor::operator()<cudf::list_view>(cudf::scalar const& lhs,
                                                         cudf::scalar const& rhs)
{
  CUDF_FAIL("Unsupported scalar compare type: list_view");
}

template <>
void compare_scalar_functor::operator()<cudf::struct_view>(cudf::scalar const& lhs,
                                                           cudf::scalar const& rhs)
{
  CUDF_FAIL("Unsupported scalar compare type: struct_view");
}

}  // anonymous namespace

void expect_scalars_equal(cudf::scalar const& lhs, cudf::scalar const& rhs)
{
  EXPECT_EQ(lhs.type(), rhs.type());
  EXPECT_EQ(lhs.is_valid(), rhs.is_valid());

  if (lhs.is_valid() && rhs.is_valid() && lhs.type() == rhs.type()) {
    type_dispatcher(lhs.type(), compare_scalar_functor{}, lhs, rhs);
  }
}

}  // namespace test
}  // namespace cudf
