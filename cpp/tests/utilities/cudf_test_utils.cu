/*
 * Copyright (c) 2018, NVIDIA CORPORATION.
 *
 * Copyright 2019 BlazingDB, Inc.
 *     Copyright 2019 Eyal Rozenberg <eyalroz@blazingdb.com>
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "cudf_test_utils.cuh"
#include <nvstrings/NVCategory.h>
#include <nvstrings/NVStrings.h>

namespace {

namespace detail {

// When streaming char-like types, the standard library streams tend to treat
// them as characters rather than numbers, e.g. you would get an 'a' instead of 97.
// The following function(s) ensure we "promote" such values to integers before
// they're streamed

template <typename T>
const T& promote_for_streaming(const T& x) { return x; }


//int promote_for_streaming(const char& x)          { return x; }
//int promote_for_streaming(const unsigned char& x) { return x; }
int promote_for_streaming(const signed char& x)   { return x; }

} // namespace detail


struct column_printer {
    template<typename Element>
    void operator()(gdf_column const* the_column, unsigned min_printing_width)
    {
        gdf_size_type num_rows { the_column->size };

        Element const* column_data { static_cast<Element const*>(the_column->data) };

        std::vector<Element> host_side_data(num_rows);
        hipMemcpy(host_side_data.data(), column_data, num_rows * sizeof(Element), hipMemcpyDeviceToHost);

        gdf_size_type const num_masks { gdf_valid_allocation_size(num_rows) };
        std::vector<gdf_valid_type> h_mask(num_masks, ~gdf_valid_type { 0 });
        if (nullptr != the_column->valid) {
            hipMemcpy(h_mask.data(), the_column->valid, num_masks * sizeof(gdf_valid_type), hipMemcpyDeviceToHost);
        }

        for (gdf_size_type i = 0; i < num_rows; ++i) {
            std::cout << std::setw(min_printing_width);
            if (gdf_is_valid(h_mask.data(), i)) {
                std::cout << detail::promote_for_streaming(host_side_data[i]);
            }
            else {
                std::cout << null_representative;
            }
            std::cout << ' ';
        }
        std::cout << std::endl;

        if(the_column->dtype == GDF_STRING_CATEGORY){
            std::cout<<"Data on category:\n";
            size_t length = 1;

            if(the_column->dtype_info.category != nullptr){
                size_t keys_size = static_cast<NVCategory *>(the_column->dtype_info.category)->keys_size();
                if(keys_size>0){
                    char ** data = new char *[keys_size];
                    for(size_t i=0; i<keys_size; i++){
                        data[i]=new char[length+1];
                    }

                    static_cast<NVCategory *>(the_column->dtype_info.category)->get_keys()->to_host(data, 0, keys_size);

                    for(size_t i=0; i<keys_size; i++){
                        data[i][length]=0;
                    }

                    for(size_t i=0; i<keys_size; i++){
                        std::cout<<"("<<data[i]<<"|"<<i<<")\t";
                    }
                    std::cout<<std::endl;
                }
            }
        }
    }
};
} // namespace

void print_gdf_column(gdf_column const * the_column, unsigned min_printing_width)
{
    cudf::type_dispatcher(the_column->dtype, column_printer{}, the_column, min_printing_width);
}

void print_valid_data(const gdf_valid_type *validity_mask,
                      const size_t num_rows)
{
  hipError_t error;
  hipPointerAttribute_t attrib;
  hipPointerGetAttributes(&attrib, validity_mask);
  error = hipGetLastError();

  std::vector<gdf_valid_type> h_mask(gdf_valid_allocation_size(num_rows));
  if (error != hipErrorInvalidValue && isDeviceType(attrib))
    hipMemcpy(h_mask.data(), validity_mask, gdf_valid_allocation_size(num_rows), hipMemcpyDeviceToHost);
  else
    memcpy(h_mask.data(), validity_mask, gdf_valid_allocation_size(num_rows));

  std::transform(
      h_mask.begin(), h_mask.begin() + gdf_num_bitmask_elements(num_rows),
      std::ostream_iterator<std::string>(std::cout, " "), [](gdf_valid_type x) {
        auto bits = std::bitset<GDF_VALID_BITSIZE>(x).to_string('@');
        return std::string(bits.rbegin(), bits.rend());
      });
  std::cout << std::endl;
}

gdf_size_type count_valid_bits_host(
    std::vector<gdf_valid_type> const& masks, gdf_size_type const num_rows)
{
  if ((0 == num_rows) || (0 == masks.size())) {
    return 0;
  }

  gdf_size_type count{0};

  // Count the valid bits for all masks except the last one
  for (gdf_size_type i = 0; i < (gdf_num_bitmask_elements(num_rows) - 1); ++i) {
    gdf_valid_type current_mask = masks[i];

    while (current_mask > 0) {
      current_mask &= (current_mask - 1);
      count++;
    }
  }

  // Only count the bits in the last mask that correspond to rows
  int num_rows_last_mask = num_rows % GDF_VALID_BITSIZE;
  if (num_rows_last_mask == 0) {
    num_rows_last_mask = GDF_VALID_BITSIZE;
  }

  // Mask off only the bits that correspond to rows
  gdf_valid_type const rows_mask = ( gdf_valid_type{1} << num_rows_last_mask ) - 1;
  gdf_valid_type last_mask = masks[gdf_num_bitmask_elements(num_rows) - 1] & rows_mask;

  while (last_mask > 0) {
    last_mask &= (last_mask - 1);
    count++;
  }

  return count;
}
