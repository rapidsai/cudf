#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2018, NVIDIA CORPORATION.
 *
 * Copyright 2019 BlazingDB, Inc.
 *     Copyright 2019 Eyal Rozenberg <eyalroz@blazingdb.com>
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "cudf_test_utils.cuh"
#include <nvstrings/NVCategory.h>
#include <nvstrings/NVStrings.h>
#include <utilities/type_dispatcher.hpp>
#include <tests/utilities/nvcategory_utils.cuh>
#include <cudf/functions.h>

namespace {

static constexpr char null_signifier = '@';

namespace detail {

// When streaming char-like types, the standard library streams tend to treat
// them as characters rather than numbers, e.g. you would get an 'a' instead of 97.
// The following function(s) ensure we "promote" such values to integers before
// they're streamed

template <typename T>
const T& promote_for_streaming(const T& x) { return x; }


//int promote_for_streaming(const char& x)          { return x; }
//int promote_for_streaming(const unsigned char& x) { return x; }
int promote_for_streaming(const signed char& x)   { return x; }

} // namespace detail


struct column_printer {
  template<typename Element>
  void operator()(gdf_column const* the_column, unsigned min_printing_width,
                  std::ostream& stream)
  {
    gdf_size_type num_rows { the_column->size };

    Element const* column_data { static_cast<Element const*>(the_column->data) };

    std::vector<Element> host_side_data(num_rows);
    hipMemcpy(host_side_data.data(), column_data, num_rows * sizeof(Element),
               hipMemcpyDeviceToHost);

    gdf_size_type const num_masks { gdf_valid_allocation_size(num_rows) };
    std::vector<gdf_valid_type> h_mask(num_masks, ~gdf_valid_type { 0 });
    if (nullptr != the_column->valid) {
      hipMemcpy(h_mask.data(), the_column->valid, num_masks * sizeof(gdf_valid_type),
                 hipMemcpyDeviceToHost);
    }

    for (gdf_size_type i = 0; i < num_rows; ++i) {
      stream << std::setw(min_printing_width);
      if (gdf_is_valid(h_mask.data(), i)) {
        stream << detail::promote_for_streaming(host_side_data[i]);
      }
      else {
        stream << null_representative;
      }
      stream << ' ';
    }
    stream << std::endl;

    if(the_column->dtype == GDF_STRING_CATEGORY){
      stream<<"Category Data (index | key):\n";

      if(the_column->dtype_info.category != nullptr){
        NVCategory *category =
          static_cast<NVCategory *>(the_column->dtype_info.category);
        
        size_t keys_size = category->keys_size();
        NVStrings *keys = category->get_keys();
        
        if (keys_size>0) {
          char ** data = new char *[keys_size];
          int * byte_sizes = new int[keys_size];
          keys->byte_count(byte_sizes, false);
          for(size_t i=0; i<keys_size; i++){
            data[i]=new char[std::max(2, byte_sizes[i])];
          }

          keys->to_host(data, 0, keys_size);

          for(size_t i=0; i<keys_size; i++){ // null terminate strings
            // TODO: nvstrings overwrites data[i] ifit is a null string
            // Update this based on resolution of https://github.com/rapidsai/custrings/issues/330
            if (byte_sizes[i]!=-1)  
              data[i][byte_sizes[i]]=0;
          }
          
          for(size_t i=0; i<keys_size; i++){ // print category strings
            stream << "(" << i << "|";
            if (data[i] == nullptr)
               stream << null_signifier; // account for null
            else
              stream << data[i];
            stream << ")\t";
          }
          stream<<std::endl;

          for(size_t i=0; i<keys_size; i++){
              delete data[i];
          }
          delete [] data;
          delete [] byte_sizes;
        }
      }
    }
  }
};

/**---------------------------------------------------------------------------*
 * @brief Functor for comparing whether two elements from two gdf_columns are
 * equal.
 *
 *---------------------------------------------------------------------------**/
template <typename T>
struct elements_equal {
  gdf_column lhs_col;
  gdf_column rhs_col;
  bool nulls_are_equivalent;

  using bit_mask_t = bit_mask::bit_mask_t;

  /**---------------------------------------------------------------------------*
   * @brief Constructs functor for comparing elements between two gdf_column's
   *
   * @param lhs The left column for comparison
   * @param rhs The right column for comparison
   * @param nulls_are_equal Desired behavior for whether or not nulls are
   * treated as equal to other nulls. Defaults to true.
   *---------------------------------------------------------------------------**/
  __host__ __device__ elements_equal(gdf_column lhs, gdf_column rhs,
                                     bool nulls_are_equal = true)
      : lhs_col{lhs}, rhs_col{rhs}, nulls_are_equivalent{nulls_are_equal} {}

  __device__ bool operator()(gdf_index_type row) {
    bool const lhs_is_valid{gdf_is_valid(lhs_col.valid, row)};
    bool const rhs_is_valid{gdf_is_valid(rhs_col.valid, row)};

    if (lhs_is_valid and rhs_is_valid) {
      return static_cast<T const*>(lhs_col.data)[row] ==
             static_cast<T const*>(rhs_col.data)[row];
    }

    // If one value is valid but the other is not
    if (lhs_is_valid != rhs_is_valid) {
      return false;
    }

    return nulls_are_equivalent;
  }
};

} // namespace anonymous

/**
 * ---------------------------------------------------------------------------*
 * @brief Compare two gdf_columns on all fields, including pairwise comparison
 * of data and valid arrays
 *
 * @tparam T The type of columns to compare
 * @param left The left column
 * @param right The right column
 * @return bool Whether or not the columns are equal
 * ---------------------------------------------------------------------------**/
template <typename T>
bool gdf_equal_columns(gdf_column const& left, gdf_column const& right)
{
  if (left.size != right.size) return false;
  if (left.dtype != right.dtype) return false;
  if (left.null_count != right.null_count) return false;
  if (left.dtype_info.time_unit != right.dtype_info.time_unit) return false;

  if ((left.col_name == nullptr) != (right.col_name == nullptr))
    return false; // if one is null but not both

  if (left.col_name != nullptr && std::strcmp(left.col_name, right.col_name) != 0)
    return false;

  if ((left.data == nullptr) != (right.data == nullptr))
    return false;  // if one is null but not both
  
  if ((left.valid == nullptr) != (right.valid == nullptr))
    return false;  // if one is null but not both

  if (left.data == nullptr)
    return true;  // logically, both are null

  if (left.dtype == GDF_STRING_CATEGORY) {
    // Transfer input column to host
    std::vector<std::string> left_data, right_data;
    std::vector<gdf_valid_type> left_bitmask, right_bitmask;
    std::tie(left_data, left_bitmask) =
      cudf::test::nvcategory_column_to_host(const_cast<gdf_column*>(&left));
    std::tie(right_data, right_bitmask) =
      cudf::test::nvcategory_column_to_host(const_cast<gdf_column*>(&right));

    CHECK_STREAM(0);

    if (left_data.size() != right_data.size())
      return false;
    
    for (size_t i = 0; i < left_data.size(); i++) {
      bool const left_is_valid{gdf_is_valid(left_bitmask.data(), i)};
      bool const right_is_valid{gdf_is_valid(right_bitmask.data(), i)};

      if (left_is_valid != right_is_valid)
        return false;
      else if (left_is_valid && (left_data[i] != right_data[i]))
        return false;
    }

    return true;
  }
  else {
    if ((left.dtype_info.category != nullptr) || (right.dtype_info.category != nullptr))
      return false;  // category must be nullptr

    bool equal_data = thrust::all_of(rmm::exec_policy()->on(0),
                                     thrust::make_counting_iterator(0),
                                     thrust::make_counting_iterator(left.size),
                                     elements_equal<T>{left, right});
    
    CHECK_STREAM(0);
  
    return equal_data;
  }
}

namespace {

struct columns_equal
{
  template <typename T>
  bool operator()(gdf_column const& left, gdf_column const& right) {
    return gdf_equal_columns<T>(left, right);
  }
};

}; // namespace anonymous

// Type-erased version of gdf_equal_columns
bool gdf_equal_columns(gdf_column const& left, gdf_column const& right)
{
  return cudf::type_dispatcher(left.dtype, columns_equal{}, left, right);
}

void print_gdf_column(gdf_column const * the_column, unsigned min_printing_width, std::ostream& stream)
{
  cudf::type_dispatcher(the_column->dtype, column_printer{}, 
                        the_column, min_printing_width, stream);
}

void print_valid_data(const gdf_valid_type *validity_mask,
                      const size_t num_rows)
{
  hipError_t error;
  hipPointerAttribute_t attrib;
  hipPointerGetAttributes(&attrib, validity_mask);
  error = hipGetLastError();

  std::vector<gdf_valid_type> h_mask(gdf_valid_allocation_size(num_rows));
  if (error != hipErrorInvalidValue && isDeviceType(attrib))
    hipMemcpy(h_mask.data(), validity_mask, gdf_valid_allocation_size(num_rows),
               hipMemcpyDeviceToHost);
  else
    memcpy(h_mask.data(), validity_mask, gdf_valid_allocation_size(num_rows));

  std::transform(
      h_mask.begin(), h_mask.begin() + gdf_num_bitmask_elements(num_rows),
      std::ostream_iterator<std::string>(std::cout, " "), [](gdf_valid_type x) {
        auto bits = std::bitset<GDF_VALID_BITSIZE>(x).to_string(null_signifier);
        return std::string(bits.rbegin(), bits.rend());
      });
  std::cout << std::endl;
}

gdf_size_type count_valid_bits_host(
    std::vector<gdf_valid_type> const& masks, gdf_size_type const num_rows)
{
  if ((0 == num_rows) || (0 == masks.size())) {
    return 0;
  }

  gdf_size_type count{0};

  // Count the valid bits for all masks except the last one
  for (gdf_size_type i = 0; i < (gdf_num_bitmask_elements(num_rows) - 1); ++i) {
    gdf_valid_type current_mask = masks[i];

    while (current_mask > 0) {
      current_mask &= (current_mask - 1);
      count++;
    }
  }

  // Only count the bits in the last mask that correspond to rows
  int num_rows_last_mask = num_rows % GDF_VALID_BITSIZE;
  if (num_rows_last_mask == 0) {
    num_rows_last_mask = GDF_VALID_BITSIZE;
  }

  // Mask off only the bits that correspond to rows
  gdf_valid_type const rows_mask = ( gdf_valid_type{1} << num_rows_last_mask ) - 1;
  gdf_valid_type last_mask = masks[gdf_num_bitmask_elements(num_rows) - 1] & rows_mask;

  while (last_mask > 0) {
    last_mask &= (last_mask - 1);
    count++;
  }

  return count;
}
