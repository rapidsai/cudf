#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <cudf/column/column.hpp>
#include <cudf/column/column_view.hpp>
#include <cudf/copying.hpp>
#include <cudf/null_mask.hpp>
#include <cudf/transform.hpp>
#include <cudf/types.hpp>
#include <cudf/utilities/type_dispatcher.hpp>
#include <tests/utilities/base_fixture.hpp>
#include <tests/utilities/column_utilities.hpp>
#include <tests/utilities/column_wrapper.hpp>
#include <tests/utilities/cudf_gtest.hpp>
#include <tests/utilities/type_list_utilities.hpp>
#include <tests/utilities/type_lists.hpp>

#include <thrust/sequence.h>
#include <random>

template <typename T>
struct TypedColumnTest : public cudf::test::BaseFixture {
  cudf::data_type type() { return cudf::data_type{cudf::type_to_id<T>()}; }

  TypedColumnTest()
    : data{_num_elements * cudf::size_of(type())},
      mask{cudf::bitmask_allocation_size_bytes(_num_elements)}
  {
    auto typed_data = static_cast<char*>(data.data());
    auto typed_mask = static_cast<char*>(mask.data());
    thrust::sequence(thrust::device, typed_data, typed_data + data.size());
    thrust::sequence(thrust::device, typed_mask, typed_mask + mask.size());
  }

  cudf::size_type num_elements() { return _num_elements; }

  std::random_device r;
  std::default_random_engine generator{r()};
  std::uniform_int_distribution<cudf::size_type> distribution{200, 1000};
  cudf::size_type _num_elements{distribution(generator)};
  rmm::device_buffer data{};
  rmm::device_buffer mask{};
  rmm::device_buffer all_valid_mask{create_null_mask(num_elements(), cudf::mask_state::ALL_VALID)};
  rmm::device_buffer all_null_mask{create_null_mask(num_elements(), cudf::mask_state::ALL_NULL)};
};

TYPED_TEST_CASE(TypedColumnTest, cudf::test::Types<int32_t>);

/**
 * @brief Verifies equality of the properties and data of a `column`'s views.
 *
 * @param col The `column` to verify
 **/
void verify_column_views(cudf::column col)
{
  cudf::column_view view                 = col;
  cudf::mutable_column_view mutable_view = col;
  EXPECT_EQ(col.type(), view.type());
  EXPECT_EQ(col.type(), mutable_view.type());
  EXPECT_EQ(col.size(), view.size());
  EXPECT_EQ(col.size(), mutable_view.size());
  EXPECT_EQ(col.null_count(), view.null_count());
  EXPECT_EQ(col.null_count(), mutable_view.null_count());
  EXPECT_EQ(col.nullable(), view.nullable());
  EXPECT_EQ(col.nullable(), mutable_view.nullable());
  EXPECT_EQ(col.num_children(), view.num_children());
  EXPECT_EQ(col.num_children(), mutable_view.num_children());
  EXPECT_EQ(view.head(), mutable_view.head());
  EXPECT_EQ(view.data<char>(), mutable_view.data<char>());
  EXPECT_EQ(view.offset(), mutable_view.offset());
}

TYPED_TEST(TypedColumnTest, DefaultNullCountNoMask)
{
  cudf::column col{this->type(), this->num_elements(), this->data};
  EXPECT_FALSE(col.nullable());
  EXPECT_FALSE(col.has_nulls());
  EXPECT_EQ(0, col.null_count());
}

TYPED_TEST(TypedColumnTest, DefaultNullCountEmptyMask)
{
  cudf::column col{this->type(), this->num_elements(), this->data, rmm::device_buffer{}};
  EXPECT_FALSE(col.nullable());
  EXPECT_FALSE(col.has_nulls());
  EXPECT_EQ(0, col.null_count());
}

TYPED_TEST(TypedColumnTest, DefaultNullCountAllValid)
{
  cudf::column col{this->type(), this->num_elements(), this->data, this->all_valid_mask};
  EXPECT_TRUE(col.nullable());
  EXPECT_FALSE(col.has_nulls());
  EXPECT_EQ(0, col.null_count());
}

TYPED_TEST(TypedColumnTest, ExplicitNullCountAllValid)
{
  cudf::column col{this->type(), this->num_elements(), this->data, this->all_valid_mask, 0};
  EXPECT_TRUE(col.nullable());
  EXPECT_FALSE(col.has_nulls());
  EXPECT_EQ(0, col.null_count());
}

TYPED_TEST(TypedColumnTest, DefaultNullCountAllNull)
{
  cudf::column col{this->type(), this->num_elements(), this->data, this->all_null_mask};
  EXPECT_TRUE(col.nullable());
  EXPECT_TRUE(col.has_nulls());
  EXPECT_EQ(this->num_elements(), col.null_count());
}

TYPED_TEST(TypedColumnTest, ExplicitNullCountAllNull)
{
  cudf::column col{
    this->type(), this->num_elements(), this->data, this->all_null_mask, this->num_elements()};
  EXPECT_TRUE(col.nullable());
  EXPECT_TRUE(col.has_nulls());
  EXPECT_EQ(this->num_elements(), col.null_count());
}

TYPED_TEST(TypedColumnTest, SetNullCountNoMask)
{
  cudf::column col{this->type(), this->num_elements(), this->data};
  EXPECT_THROW(col.set_null_count(1), cudf::logic_error);
}

TYPED_TEST(TypedColumnTest, SetEmptyNullMaskNonZeroNullCount)
{
  cudf::column col{this->type(), this->num_elements(), this->data};
  rmm::device_buffer empty_null_mask{};
  EXPECT_THROW(col.set_null_mask(empty_null_mask, this->num_elements()), cudf::logic_error);
}

TYPED_TEST(TypedColumnTest, SetInvalidSizeNullMaskNonZeroNullCount)
{
  cudf::column col{this->type(), this->num_elements(), this->data};
  auto invalid_size_null_mask =
    create_null_mask(std::min(this->num_elements() - 50, 0), cudf::mask_state::ALL_VALID);
  EXPECT_THROW(col.set_null_mask(invalid_size_null_mask, this->num_elements()), cudf::logic_error);
}

TYPED_TEST(TypedColumnTest, SetNullCountEmptyMask)
{
  cudf::column col{this->type(), this->num_elements(), this->data, rmm::device_buffer{}};
  EXPECT_THROW(col.set_null_count(1), cudf::logic_error);
}

TYPED_TEST(TypedColumnTest, SetNullCountAllValid)
{
  cudf::column col{this->type(), this->num_elements(), this->data, this->all_valid_mask};
  EXPECT_NO_THROW(col.set_null_count(0));
  EXPECT_EQ(0, col.null_count());
}

TYPED_TEST(TypedColumnTest, SetNullCountAllNull)
{
  cudf::column col{this->type(), this->num_elements(), this->data, this->all_null_mask};
  EXPECT_NO_THROW(col.set_null_count(this->num_elements()));
  EXPECT_EQ(this->num_elements(), col.null_count());
}

TYPED_TEST(TypedColumnTest, ResetNullCountAllNull)
{
  cudf::column col{this->type(), this->num_elements(), this->data, this->all_null_mask};

  EXPECT_EQ(this->num_elements(), col.null_count());
  EXPECT_NO_THROW(col.set_null_count(cudf::UNKNOWN_NULL_COUNT));
  EXPECT_EQ(this->num_elements(), col.null_count());
}

TYPED_TEST(TypedColumnTest, ResetNullCountAllValid)
{
  cudf::column col{this->type(), this->num_elements(), this->data, this->all_valid_mask};
  EXPECT_EQ(0, col.null_count());
  EXPECT_NO_THROW(col.set_null_count(cudf::UNKNOWN_NULL_COUNT));
  EXPECT_EQ(0, col.null_count());
}

TYPED_TEST(TypedColumnTest, CopyDataNoMask)
{
  cudf::column col{this->type(), this->num_elements(), this->data};
  EXPECT_EQ(this->type(), col.type());
  EXPECT_FALSE(col.nullable());
  EXPECT_EQ(0, col.null_count());
  EXPECT_EQ(this->num_elements(), col.size());
  EXPECT_EQ(0, col.num_children());

  verify_column_views(col);

  // Verify deep copy
  cudf::column_view v = col;
  EXPECT_NE(v.head(), this->data.data());
  CUDF_TEST_EXPECT_EQUAL_BUFFERS(v.head(), this->data.data(), this->data.size());
}

TYPED_TEST(TypedColumnTest, MoveDataNoMask)
{
  void* original_data = this->data.data();
  cudf::column col{this->type(), this->num_elements(), std::move(this->data)};
  EXPECT_EQ(this->type(), col.type());
  EXPECT_FALSE(col.nullable());
  EXPECT_EQ(0, col.null_count());
  EXPECT_EQ(this->num_elements(), col.size());
  EXPECT_EQ(0, col.num_children());

  verify_column_views(col);

  // Verify shallow copy
  cudf::column_view v = col;
  EXPECT_EQ(v.head(), original_data);
}

TYPED_TEST(TypedColumnTest, CopyDataAndMask)
{
  cudf::column col{this->type(), this->num_elements(), this->data, this->all_valid_mask};
  EXPECT_EQ(this->type(), col.type());
  EXPECT_TRUE(col.nullable());
  EXPECT_EQ(0, col.null_count());
  EXPECT_EQ(this->num_elements(), col.size());
  EXPECT_EQ(0, col.num_children());

  verify_column_views(col);

  // Verify deep copy
  cudf::column_view v = col;
  EXPECT_NE(v.head(), this->data.data());
  EXPECT_NE(v.null_mask(), this->all_valid_mask.data());
  CUDF_TEST_EXPECT_EQUAL_BUFFERS(v.head(), this->data.data(), this->data.size());
  CUDF_TEST_EXPECT_EQUAL_BUFFERS(v.null_mask(), this->all_valid_mask.data(), this->mask.size());
}

TYPED_TEST(TypedColumnTest, MoveDataAndMask)
{
  void* original_data = this->data.data();
  void* original_mask = this->all_valid_mask.data();
  cudf::column col{
    this->type(), this->num_elements(), std::move(this->data), std::move(this->all_valid_mask)};
  EXPECT_EQ(this->type(), col.type());
  EXPECT_TRUE(col.nullable());
  EXPECT_EQ(0, col.null_count());
  EXPECT_EQ(this->num_elements(), col.size());
  EXPECT_EQ(0, col.num_children());

  verify_column_views(col);

  // Verify shallow copy
  cudf::column_view v = col;
  EXPECT_EQ(v.head(), original_data);
  EXPECT_EQ(v.null_mask(), original_mask);
}

TYPED_TEST(TypedColumnTest, CopyConstructorNoMask)
{
  cudf::column original{this->type(), this->num_elements(), this->data};
  cudf::column copy{original};
  verify_column_views(copy);
  CUDF_TEST_EXPECT_COLUMNS_EQUAL(original, copy);

  // Verify deep copy
  cudf::column_view original_view = original;
  cudf::column_view copy_view     = copy;
  EXPECT_NE(original_view.head(), copy_view.head());
}

TYPED_TEST(TypedColumnTest, CopyConstructorWithMask)
{
  cudf::column original{this->type(), this->num_elements(), this->data, this->all_valid_mask};
  cudf::column copy{original};
  verify_column_views(copy);
  CUDF_TEST_EXPECT_COLUMNS_EQUAL(original, copy);

  // Verify deep copy
  cudf::column_view original_view = original;
  cudf::column_view copy_view     = copy;
  EXPECT_NE(original_view.head(), copy_view.head());
  EXPECT_NE(original_view.null_mask(), copy_view.null_mask());
}

TYPED_TEST(TypedColumnTest, MoveConstructorNoMask)
{
  cudf::column original{this->type(), this->num_elements(), this->data};

  auto original_data = original.view().head();

  cudf::column moved_to{std::move(original)};

  EXPECT_EQ(0, original.size());
  EXPECT_EQ(cudf::data_type{cudf::type_id::EMPTY}, original.type());

  verify_column_views(moved_to);

  // Verify move
  cudf::column_view moved_to_view = moved_to;
  EXPECT_EQ(original_data, moved_to_view.head());
}

TYPED_TEST(TypedColumnTest, MoveConstructorWithMask)
{
  cudf::column original{this->type(), this->num_elements(), this->data, this->all_valid_mask};
  auto original_data = original.view().head();
  auto original_mask = original.view().null_mask();
  cudf::column moved_to{std::move(original)};
  verify_column_views(moved_to);

  EXPECT_EQ(0, original.size());
  EXPECT_EQ(cudf::data_type{cudf::type_id::EMPTY}, original.type());

  // Verify move
  cudf::column_view moved_to_view = moved_to;
  EXPECT_EQ(original_data, moved_to_view.head());
  EXPECT_EQ(original_mask, moved_to_view.null_mask());
}

TYPED_TEST(TypedColumnTest, ConstructWithChildren)
{
  std::vector<std::unique_ptr<cudf::column>> children;
  children.emplace_back(std::make_unique<cudf::column>(
    cudf::data_type{cudf::type_id::INT8}, 42, this->data, this->all_valid_mask));
  children.emplace_back(std::make_unique<cudf::column>(
    cudf::data_type{cudf::type_id::FLOAT64}, 314, this->data, this->all_valid_mask));
  cudf::column col{this->type(),
                   this->num_elements(),
                   this->data,
                   this->all_valid_mask,
                   cudf::UNKNOWN_NULL_COUNT,
                   std::move(children)};

  verify_column_views(col);
  EXPECT_EQ(2, col.num_children());
  EXPECT_EQ(cudf::data_type{cudf::type_id::INT8}, col.child(0).type());
  EXPECT_EQ(42, col.child(0).size());
  EXPECT_EQ(cudf::data_type{cudf::type_id::FLOAT64}, col.child(1).type());
  EXPECT_EQ(314, col.child(1).size());
}

TYPED_TEST(TypedColumnTest, ReleaseNoChildren)
{
  cudf::column col{this->type(), this->num_elements(), this->data, this->all_valid_mask};
  auto original_data = col.view().head();
  auto original_mask = col.view().null_mask();

  cudf::column::contents contents = col.release();
  EXPECT_EQ(original_data, contents.data->data());
  EXPECT_EQ(original_mask, contents.null_mask->data());
  EXPECT_EQ(0u, contents.children.size());
  EXPECT_EQ(0, col.size());
  EXPECT_EQ(0, col.null_count());
  EXPECT_EQ(cudf::data_type{cudf::type_id::EMPTY}, col.type());
  EXPECT_EQ(0, col.num_children());
}

TYPED_TEST(TypedColumnTest, ReleaseWithChildren)
{
  std::vector<std::unique_ptr<cudf::column>> children;
  children.emplace_back(std::make_unique<cudf::column>(
    this->type(), this->num_elements(), this->data, this->all_valid_mask));
  children.emplace_back(std::make_unique<cudf::column>(
    this->type(), this->num_elements(), this->data, this->all_valid_mask));
  cudf::column col{this->type(),
                   this->num_elements(),
                   this->data,
                   this->all_valid_mask,
                   cudf::UNKNOWN_NULL_COUNT,
                   std::move(children)};

  auto original_data = col.view().head();
  auto original_mask = col.view().null_mask();

  cudf::column::contents contents = col.release();
  EXPECT_EQ(original_data, contents.data->data());
  EXPECT_EQ(original_mask, contents.null_mask->data());
  EXPECT_EQ(2u, contents.children.size());
  EXPECT_EQ(0, col.size());
  EXPECT_EQ(0, col.null_count());
  EXPECT_EQ(cudf::data_type{cudf::type_id::EMPTY}, col.type());
  EXPECT_EQ(0, col.num_children());
}

TYPED_TEST(TypedColumnTest, ColumnViewConstructorWithMask)
{
  cudf::column original{this->type(), this->num_elements(), this->data, this->all_valid_mask};
  cudf::column_view original_view = original;
  cudf::column copy{original_view};
  verify_column_views(copy);
  CUDF_TEST_EXPECT_COLUMNS_EQUAL(original, copy);

  // Verify deep copy
  cudf::column_view copy_view = copy;
  EXPECT_NE(original_view.head(), copy_view.head());
  EXPECT_NE(original_view.null_mask(), copy_view.null_mask());
}

template <typename T>
struct ListsColumnTest : public cudf::test::BaseFixture {
};

using NumericTypesNotBool =
  cudf::test::Concat<cudf::test::IntegralTypesNotBool, cudf::test::FloatingPointTypes>;

TYPED_TEST_CASE(ListsColumnTest, NumericTypesNotBool);

TYPED_TEST(ListsColumnTest, ListsColumnViewConstructor)
{
  cudf::test::lists_column_wrapper<TypeParam> list{{1, 2}, {3, 4}, {5, 6, 7}, {8, 9}};

  auto result = std::make_unique<cudf::column>(list);

  cudf::test::expect_columns_equal(list, result->view());
}

TYPED_TEST(ListsColumnTest, ListsSlicedColumnViewConstructor)
{
  cudf::test::lists_column_wrapper<TypeParam> list{{1, 2}, {3, 4}, {5, 6, 7}, {8, 9}};
  cudf::test::lists_column_wrapper<TypeParam> expect{{3, 4}, {5, 6, 7}};

  auto sliced = cudf::slice(list, {1, 3}).front();
  auto result = std::make_unique<cudf::column>(sliced);

  cudf::test::expect_columns_equal(expect, result->view());
}

TYPED_TEST(ListsColumnTest, ListsSlicedColumnViewConstructorWithNulls)
{
  auto valids = cudf::test::make_counting_transform_iterator(
    0, [](auto i) { return i % 2 == 0 ? true : false; });

  auto expect_valids = cudf::test::make_counting_transform_iterator(
    0, [](auto i) { return i % 2 == 0 ? false : true; });

  using LCW = cudf::test::lists_column_wrapper<TypeParam>;

  cudf::test::lists_column_wrapper<TypeParam> list{
    {{{{1, 2}, {3, 4}}, valids}, LCW{}, {{{5, 6, 7}, LCW{}, {8, 9}}, valids}, LCW{}, LCW{}},
    valids};

  cudf::test::lists_column_wrapper<TypeParam> expect{
    {LCW{}, {{{5, 6, 7}, LCW{}, {8, 9}}, valids}, LCW{}, LCW{}}, expect_valids};

  auto sliced = cudf::slice(list, {1, 5}).front();
  auto result = std::make_unique<cudf::column>(sliced);

  cudf::test::expect_columns_equal(expect, result->view());

  // TODO: null mask equality is being checked separately because
  // expect_columns_equal doesn't do the check for lists columns.
  // This is fixed in https://github.com/rapidsai/cudf/pull/5904,
  // so we should remove this check after that's merged:
  cudf::test::expect_columns_equal(
    cudf::mask_to_bools(result->view().null_mask(), 0, 4)->view(),
    cudf::mask_to_bools(static_cast<cudf::column_view>(expect).null_mask(), 0, 4)->view());
}

CUDF_TEST_PROGRAM_MAIN()
