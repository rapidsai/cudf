/*
 * Copyright (c) 2019, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <cudf/column/column.hpp>
#include <cudf/column/column_view.hpp>
#include <cudf/null_mask.hpp>
#include <cudf/types.hpp>
#include <cudf/utilities/type_dispatcher.hpp>
#include <tests/utilities/base_fixture.hpp>
#include <tests/utilities/column_utilities.hpp>
#include <tests/utilities/cudf_gtest.hpp>
#include <tests/utilities/type_list_utilities.hpp>
#include <tests/utilities/type_lists.hpp>

#include <thrust/sequence.h>
#include <random>

#include <gmock/gmock.h>

template <typename T>
struct TypedColumnTest : public cudf::test::BaseFixture {
  static std::size_t data_size() { return 1000; }
  static std::size_t mask_size() { return 100; }
  cudf::data_type type() { return cudf::data_type{cudf::experimental::type_to_id<T>()}; }

  TypedColumnTest()
      : data{_num_elements * cudf::size_of(type())},
        mask{cudf::bitmask_allocation_size_bytes(_num_elements)} {
    auto typed_data = static_cast<char*>(data.data());
    auto typed_mask = static_cast<char*>(mask.data());
    thrust::sequence(thrust::device, typed_data, typed_data + data_size());
    thrust::sequence(thrust::device, typed_mask, typed_mask + mask_size());
  }

  cudf::size_type num_elements() { return _num_elements; }

  std::random_device r;
  std::default_random_engine generator{r()};
  std::uniform_int_distribution<cudf::size_type> distribution{200, 1000};
  cudf::size_type _num_elements{distribution(generator)};
  rmm::device_buffer data{};
  rmm::device_buffer mask{};
  rmm::device_buffer all_valid_mask{
      create_null_mask(num_elements(), cudf::mask_state::ALL_VALID)};
  rmm::device_buffer all_null_mask{
      create_null_mask(num_elements(), cudf::mask_state::ALL_NULL)};
};

TYPED_TEST_CASE(TypedColumnTest, cudf::test::Types<int32_t>);

/**---------------------------------------------------------------------------*
 * @brief Verifies equality of the properties and data of a `column`'s views.
 *
 * @param col The `column` to verify
 *---------------------------------------------------------------------------**/
void verify_column_views(cudf::column col) {
  cudf::column_view view = col;
  cudf::mutable_column_view mutable_view = col;
  EXPECT_EQ(col.type(), view.type());
  EXPECT_EQ(col.type(), mutable_view.type());
  EXPECT_EQ(col.size(), view.size());
  EXPECT_EQ(col.size(), mutable_view.size());
  EXPECT_EQ(col.null_count(), view.null_count());
  EXPECT_EQ(col.null_count(), mutable_view.null_count());
  EXPECT_EQ(col.nullable(), view.nullable());
  EXPECT_EQ(col.nullable(), mutable_view.nullable());
  EXPECT_EQ(col.num_children(), view.num_children());
  EXPECT_EQ(col.num_children(), mutable_view.num_children());
  EXPECT_EQ(view.head(), mutable_view.head());
  EXPECT_EQ(view.data<char>(), mutable_view.data<char>());
  EXPECT_EQ(view.offset(), mutable_view.offset());
}

TYPED_TEST(TypedColumnTest, DefaultNullCountNoMask) {
  cudf::column col{this->type(), this->num_elements(), this->data};
  EXPECT_FALSE(col.nullable());
  EXPECT_FALSE(col.has_nulls());
  EXPECT_EQ(0, col.null_count());
}

TYPED_TEST(TypedColumnTest, DefaultNullCountEmptyMask) {
  cudf::column col{this->type(), this->num_elements(), this->data,
                   rmm::device_buffer{}};
  EXPECT_FALSE(col.nullable());
  EXPECT_FALSE(col.has_nulls());
  EXPECT_EQ(0, col.null_count());
}

TYPED_TEST(TypedColumnTest, DefaultNullCountAllValid) {
  cudf::column col{this->type(), this->num_elements(), this->data,
                   this->all_valid_mask};
  EXPECT_TRUE(col.nullable());
  EXPECT_FALSE(col.has_nulls());
  EXPECT_EQ(0, col.null_count());
}

TYPED_TEST(TypedColumnTest, ExplicitNullCountAllValid) {
  cudf::column col{this->type(), this->num_elements(), this->data,
                   this->all_valid_mask, 0};
  EXPECT_TRUE(col.nullable());
  EXPECT_FALSE(col.has_nulls());
  EXPECT_EQ(0, col.null_count());
}

TYPED_TEST(TypedColumnTest, DefaultNullCountAllNull) {
  cudf::column col{this->type(), this->num_elements(), this->data,
                   this->all_null_mask};
  EXPECT_TRUE(col.nullable());
  EXPECT_TRUE(col.has_nulls());
  EXPECT_EQ(this->num_elements(), col.null_count());
}

TYPED_TEST(TypedColumnTest, ExplicitNullCountAllNull) {
  cudf::column col{this->type(), this->num_elements(), this->data,
                   this->all_null_mask, this->num_elements()};
  EXPECT_TRUE(col.nullable());
  EXPECT_TRUE(col.has_nulls());
  EXPECT_EQ(this->num_elements(), col.null_count());
}

TYPED_TEST(TypedColumnTest, SetNullCountNoMask) {
  cudf::column col{this->type(), this->num_elements(), this->data};
  EXPECT_THROW(col.set_null_count(1), cudf::logic_error);
}

TYPED_TEST(TypedColumnTest, SetNullCountEmptyMask) {
  cudf::column col{this->type(), this->num_elements(), this->data,
                   rmm::device_buffer{}};
  EXPECT_THROW(col.set_null_count(1), cudf::logic_error);
}

TYPED_TEST(TypedColumnTest, SetNullCountAllValid) {
  cudf::column col{this->type(), this->num_elements(), this->data,
                   this->all_valid_mask};
  EXPECT_NO_THROW(col.set_null_count(0));
  EXPECT_EQ(0, col.null_count());
}

TYPED_TEST(TypedColumnTest, SetNullCountAllNull) {
  cudf::column col{this->type(), this->num_elements(), this->data,
                   this->all_null_mask};
  EXPECT_NO_THROW(col.set_null_count(this->num_elements()));
  EXPECT_EQ(this->num_elements(), col.null_count());
}

TYPED_TEST(TypedColumnTest, ResetNullCountAllNull) {
  cudf::column col{this->type(), this->num_elements(), this->data,
                   this->all_null_mask};

  EXPECT_EQ(this->num_elements(), col.null_count());
  EXPECT_NO_THROW(col.set_null_count(cudf::UNKNOWN_NULL_COUNT));
  EXPECT_EQ(this->num_elements(), col.null_count());
}

TYPED_TEST(TypedColumnTest, ResetNullCountAllValid) {
  cudf::column col{this->type(), this->num_elements(), this->data,
                   this->all_valid_mask};
  EXPECT_EQ(0, col.null_count());
  EXPECT_NO_THROW(col.set_null_count(cudf::UNKNOWN_NULL_COUNT));
  EXPECT_EQ(0, col.null_count());
}

TYPED_TEST(TypedColumnTest, CopyDataNoMask) {
  cudf::column col{this->type(), this->num_elements(), this->data};
  EXPECT_EQ(this->type(), col.type());
  EXPECT_FALSE(col.nullable());
  EXPECT_EQ(0, col.null_count());
  EXPECT_EQ(this->num_elements(), col.size());
  EXPECT_EQ(0, col.num_children());

  verify_column_views(col);

  // Verify deep copy
  cudf::column_view v = col;
  EXPECT_NE(v.head(), this->data.data());
  cudf::test::expect_equal_buffers(v.head(), this->data.data(),
                                   this->data.size());
}

TYPED_TEST(TypedColumnTest, MoveDataNoMask) {
  void* original_data = this->data.data();
  cudf::column col{this->type(), this->num_elements(), std::move(this->data)};
  EXPECT_EQ(this->type(), col.type());
  EXPECT_FALSE(col.nullable());
  EXPECT_EQ(0, col.null_count());
  EXPECT_EQ(this->num_elements(), col.size());
  EXPECT_EQ(0, col.num_children());

  verify_column_views(col);

  // Verify shallow copy
  cudf::column_view v = col;
  EXPECT_EQ(v.head(), original_data);
}

TYPED_TEST(TypedColumnTest, CopyDataAndMask) {
  cudf::column col{this->type(), this->num_elements(), this->data,
                   this->all_valid_mask};
  EXPECT_EQ(this->type(), col.type());
  EXPECT_TRUE(col.nullable());
  EXPECT_EQ(0, col.null_count());
  EXPECT_EQ(this->num_elements(), col.size());
  EXPECT_EQ(0, col.num_children());

  verify_column_views(col);

  // Verify deep copy
  cudf::column_view v = col;
  EXPECT_NE(v.head(), this->data.data());
  EXPECT_NE(v.null_mask(), this->all_valid_mask.data());
  cudf::test::expect_equal_buffers(v.head(), this->data.data(),
                                   this->data.size());
  cudf::test::expect_equal_buffers(v.null_mask(), this->all_valid_mask.data(),
                                   this->mask.size());
}

TYPED_TEST(TypedColumnTest, MoveDataAndMask) {
  void* original_data = this->data.data();
  void* original_mask = this->all_valid_mask.data();
  cudf::column col{this->type(), this->num_elements(), std::move(this->data),
                   std::move(this->all_valid_mask)};
  EXPECT_EQ(this->type(), col.type());
  EXPECT_TRUE(col.nullable());
  EXPECT_EQ(0, col.null_count());
  EXPECT_EQ(this->num_elements(), col.size());
  EXPECT_EQ(0, col.num_children());

  verify_column_views(col);

  // Verify shallow copy
  cudf::column_view v = col;
  EXPECT_EQ(v.head(), original_data);
  EXPECT_EQ(v.null_mask(), original_mask);
}

TYPED_TEST(TypedColumnTest, CopyConstructorNoMask) {
  cudf::column original{this->type(), this->num_elements(), this->data};
  cudf::column copy{original};
  verify_column_views(copy);
  cudf::test::expect_columns_equal(original, copy);

  // Verify deep copy
  cudf::column_view original_view = original;
  cudf::column_view copy_view = copy;
  EXPECT_NE(original_view.head(), copy_view.head());
}

TYPED_TEST(TypedColumnTest, CopyConstructorWithMask) {
  cudf::column original{this->type(), this->num_elements(), this->data,
                        this->all_valid_mask};
  cudf::column copy{original};
  verify_column_views(copy);
  cudf::test::expect_columns_equal(original, copy);

  // Verify deep copy
  cudf::column_view original_view = original;
  cudf::column_view copy_view = copy;
  EXPECT_NE(original_view.head(), copy_view.head());
  EXPECT_NE(original_view.null_mask(), copy_view.null_mask());
}

TYPED_TEST(TypedColumnTest, MoveConstructorNoMask) {
  cudf::column original{this->type(), this->num_elements(), this->data};

  auto original_data = original.view().head();

  cudf::column moved_to{std::move(original)};

  EXPECT_EQ(0, original.size());
  EXPECT_EQ(cudf::data_type{cudf::type_id::EMPTY}, original.type());

  verify_column_views(moved_to);

  // Verify move
  cudf::column_view moved_to_view = moved_to;
  EXPECT_EQ(original_data, moved_to_view.head());
}

TYPED_TEST(TypedColumnTest, MoveConstructorWithMask) {
  cudf::column original{this->type(), this->num_elements(), this->data,
                        this->all_valid_mask};
  auto original_data = original.view().head();
  auto original_mask = original.view().null_mask();
  cudf::column moved_to{std::move(original)};
  verify_column_views(moved_to);

  EXPECT_EQ(0, original.size());
  EXPECT_EQ(cudf::data_type{cudf::type_id::EMPTY}, original.type());

  // Verify move
  cudf::column_view moved_to_view = moved_to;
  EXPECT_EQ(original_data, moved_to_view.head());
  EXPECT_EQ(original_mask, moved_to_view.null_mask());
}
