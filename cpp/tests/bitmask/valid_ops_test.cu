#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2018, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <tests/utilities/cudf_test_utils.cuh>
#include <tests/utilities/cudf_test_fixtures.h>

#include <cudf.h>

#include <gtest/gtest.h>
#include <gmock/gmock.h>

#include <hip/hip_runtime_api.h>

#include <chrono>

struct ValidsTest : public GdfTest {};

TEST_F(ValidsTest, NoValids)
{
  const int num_rows = 100;
  std::vector<int> data(num_rows);
  const int num_masks = std::ceil(num_rows/static_cast<float>(8));
  std::vector<gdf_valid_type> valid(num_masks,0x00);

  auto input_gdf_col = create_gdf_column(data, valid);

  int count{-1};
  gdf_error error_code = gdf_count_nonzero_mask(input_gdf_col->valid, num_rows, &count);

  ASSERT_EQ(GDF_SUCCESS,error_code) << "GDF Operation did not complete successfully.";

  EXPECT_EQ(0, count);
}

TEST_F(ValidsTest, NullValids)
{
  int count{-1};
  const gdf_size_type size{100};
  gdf_error error_code = gdf_count_nonzero_mask(nullptr, size, &count);

  ASSERT_EQ(GDF_SUCCESS,error_code) << "GDF Operation did not complete successfully.";
  EXPECT_EQ(size, count);
}

TEST_F(ValidsTest, NullCount)
{
  std::vector<int> data(0);
  std::vector<gdf_valid_type> valid{0x0};
  auto input_gdf_col = create_gdf_column(data, valid);
  gdf_error error_code = gdf_count_nonzero_mask(input_gdf_col->valid, 1, nullptr);

  ASSERT_EQ(GDF_DATASET_EMPTY,error_code) << "Expected failure for null input.";
}

TEST_F(ValidsTest, FirstRowValid)
{
  std::vector<int> data(4);
  std::vector<gdf_valid_type> valid{0x1};

  auto input_gdf_col = create_gdf_column(data, valid);

  int count{-1};
  gdf_error error_code = gdf_count_nonzero_mask(input_gdf_col->valid, 1, &count);

  ASSERT_EQ(GDF_SUCCESS,error_code) << "GDF Operation did not complete successfully.";

  EXPECT_EQ(1, count);
}

TEST_F(ValidsTest, EightRowsValid)
{
  std::vector<int> data(8);
  std::vector<gdf_valid_type> valid{0xFF};

  auto input_gdf_col = create_gdf_column(data, valid);

  int count{-1};
  gdf_error error_code = gdf_count_nonzero_mask(input_gdf_col->valid, 8, &count);

  ASSERT_EQ(GDF_SUCCESS,error_code) << "GDF Operation did not complete successfully.";

  EXPECT_EQ(8, count);
}

TEST_F(ValidsTest, EveryOtherBit)
{
  std::vector<int> data(8);
  std::vector<gdf_valid_type> valid{0xAA};

  auto input_gdf_col = create_gdf_column(data, valid);

  int count{-1};
  gdf_error error_code = gdf_count_nonzero_mask(input_gdf_col->valid, 8, &count);

  ASSERT_EQ(GDF_SUCCESS,error_code) << "GDF Operation did not complete successfully.";

  EXPECT_EQ(4, count);
}

TEST_F(ValidsTest, OtherEveryOtherBit)
{
  std::vector<int> data(8);
  std::vector<gdf_valid_type> valid{0x55};

  auto input_gdf_col = create_gdf_column(data, valid);

  int count{-1};
  gdf_error error_code = gdf_count_nonzero_mask(input_gdf_col->valid, 8, &count);

  ASSERT_EQ(GDF_SUCCESS,error_code) << "GDF Operation did not complete successfully.";

  EXPECT_EQ(4, count);
}

TEST_F(ValidsTest, 15rows)
{
  const int num_rows = 15;
  std::vector<int> data(num_rows);
  const int num_masks = std::ceil(num_rows/static_cast<float>(8));
  std::vector<gdf_valid_type> valid(num_masks,0x01);

  auto input_gdf_col = create_gdf_column(data, valid);

  int count{-1};
  gdf_error error_code = gdf_count_nonzero_mask(input_gdf_col->valid, num_rows, &count);

  ASSERT_EQ(GDF_SUCCESS,error_code) << "GDF Operation did not complete successfully.";

  EXPECT_EQ(2, count);
}

TEST_F(ValidsTest, 5rows)
{
  const int num_rows = 5;
  std::vector<int> data(num_rows);
  const int num_masks = std::ceil(num_rows/static_cast<float>(8));
  std::vector<gdf_valid_type> valid(num_masks,0x01);

  auto input_gdf_col = create_gdf_column(data, valid);

  int count{-1};
  gdf_error error_code = gdf_count_nonzero_mask(input_gdf_col->valid, num_rows, &count);

  ASSERT_EQ(GDF_SUCCESS,error_code) << "GDF Operation did not complete successfully.";

  EXPECT_EQ(1, count);
}

TEST_F(ValidsTest, 10ValidRows)
{
  const int num_rows = 10;
  std::vector<float> data(num_rows);
  const int num_masks = std::ceil(num_rows/static_cast<float>(8));
  std::vector<gdf_valid_type> valid(num_masks,0xFF);

  auto input_gdf_col = create_gdf_column(data, valid);

  int count{-1};
  gdf_error error_code = gdf_count_nonzero_mask(input_gdf_col->valid, num_rows, &count);

  ASSERT_EQ(GDF_SUCCESS,error_code) << "GDF Operation did not complete successfully.";

  EXPECT_EQ(10, count);
}

TEST_F(ValidsTest, MultipleOfEight)
{
  const int num_rows = 1024;
  std::vector<int> data(num_rows);

  const int num_masks = std::ceil(num_rows/static_cast<float>(8));
  std::vector<gdf_valid_type> valid(num_masks,0x01);

  auto input_gdf_col = create_gdf_column(data, valid);

  int count{-1};
  gdf_error error_code = gdf_count_nonzero_mask(input_gdf_col->valid, num_rows, &count);

  ASSERT_EQ(GDF_SUCCESS,error_code) << "GDF Operation did not complete successfully.";

  EXPECT_EQ(128, count);
}

TEST_F(ValidsTest, NotMultipleOfEight)
{
  const int num_rows = 1023;
  std::vector<int> data(num_rows);

  const int num_masks = std::ceil(num_rows/static_cast<float>(8));
  std::vector<gdf_valid_type> valid(num_masks, 0x80);

  auto input_gdf_col = create_gdf_column(data, valid);

  int count{-1};
  gdf_error error_code = gdf_count_nonzero_mask(input_gdf_col->valid, num_rows, &count);

  ASSERT_EQ(GDF_SUCCESS,error_code) << "GDF Operation did not complete successfully.";

  EXPECT_EQ(127, count);
}

TEST_F(ValidsTest, TenThousandRows)
{
  const int num_rows = 10000;
  std::vector<int> data(num_rows);

  const int num_masks = std::ceil(num_rows/static_cast<float>(8));
  std::vector<gdf_valid_type> valid(num_masks, 0xFF);

  auto input_gdf_col = create_gdf_column(data, valid);

  int count{-1};
  gdf_error error_code = gdf_count_nonzero_mask(input_gdf_col->valid, num_rows, &count);

  ASSERT_EQ(GDF_SUCCESS,error_code) << "GDF Operation did not complete successfully.";

  EXPECT_EQ(10000, count);
}

TEST_F(ValidsTest, PerformanceTest)
{
  const int num_rows = 100000000;
  std::vector<int> data(num_rows);

  const int num_masks = std::ceil(num_rows/8);
  std::vector<gdf_valid_type> valid(num_masks, 0x55);

  auto input_gdf_col = create_gdf_column(data, valid);

  int count{-1};
  
  auto start = std::chrono::system_clock::now();
  hipProfilerStart();
  for(int i = 0; i < 1000; ++i)
    gdf_error error_code = gdf_count_nonzero_mask(input_gdf_col->valid, num_rows, &count);
  hipProfilerStop();
  auto end = std::chrono::system_clock::now();
  std::chrono::duration<double> elapsed_seconds = end-start;
  std::cout << "Elapsed time (ms): " << elapsed_seconds.count()*1000 << std::endl;
}



