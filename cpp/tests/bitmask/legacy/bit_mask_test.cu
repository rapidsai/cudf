#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2018, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <tests/utilities/cudf_test_utils.cuh>
#include <tests/utilities/cudf_test_fixtures.h>
#include <bitmask/legacy/bit_mask.cuh>
#include <bitmask/legacy/BitMask.cuh>

#include <gtest/gtest.h>
#include <gmock/gmock.h>

#include <hip/hip_runtime_api.h>

#include <gtest/gtest.h>

#include <chrono>


struct BitMaskTest : public GdfTest {};

//
//  Kernel to count bits set in the bit mask
//
__global__ void count_bits_g(int *counter, BitMask bits) {
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
    
  int local_counter = 0;
  int i;

  for (i = index ; i < (bits.num_elements() - 1) ; i += stride) {
    local_counter += __popc(bits.get_element_device(i));
  }

  if (i == (bits.num_elements() - 1)) {
    //
    //  Special case... last word is only partial
    //
    int bits_used = bits.length() % bit_mask::bits_per_element;
    if (bits_used == 0) {
      //
      //  The whole word is used
      //
      local_counter += __popc(bits.get_element_device(i));
    } else {
      local_counter += __popc(bits.get_element_device(i) & ((bit_mask_t{1} << bits_used) - 1));
    }
  }

  atomicAdd(counter, local_counter);
}

//
//  Testing function, will set a bit in a container.  This assumes <1,1>
//  for simplicity - all of the tests are small.
//
__global__ void set_bit(gdf_size_type bit, BitMask bits) {
  bits.set_bit_unsafe(bit);
}

//
//  Kernel to do safe bit set/clear
//
__global__ void test_safe_set_clear_g(BitMask bits) {
  int index = threadIdx.x;

  if ((index % 2) == 0) {
    for (int i = index ; i < bits.length() ; i += bit_mask::bits_per_element) {
      bits.set_bit(i);
    }
  }

  for (int i = index ; i < bits.length() ; i += bit_mask::bits_per_element) {
    bits.clear_bit(i);
  }

  if ((index % 2) == 0) {
    for (int i = index ; i < bits.length() ; i += bit_mask::bits_per_element) {
      bits.set_bit(i);
    }
  }
}


__host__ gdf_error count_bits(gdf_size_type *count, const BitMask &bit_mask, int a = 1, int b = 1) {
  int *count_d;
  CUDA_TRY(hipMalloc(&count_d, sizeof(int)));
  CUDA_TRY(hipMemset(count_d, 0, sizeof(int)));
  
  count_bits_g<<<a,b>>>(count_d, bit_mask);

  CUDA_TRY(hipMemcpy(count, count_d, sizeof(int), hipMemcpyDeviceToHost));
  CUDA_TRY(hipFree(count_d));

  return GDF_SUCCESS;
}


TEST_F(BitMaskTest, NoValids)
{
  const int num_rows = 100;

  bit_mask_t *bits = nullptr;
  EXPECT_EQ(GDF_SUCCESS, bit_mask::create_bit_mask(&bits, num_rows, 0));

  BitMask bit_mask(bits, num_rows);

  gdf_size_type local_count = 0;
  EXPECT_EQ(GDF_SUCCESS, count_bits(&local_count, bit_mask));

  EXPECT_EQ(GDF_SUCCESS, bit_mask::destroy_bit_mask(bits));

  EXPECT_EQ(gdf_size_type{0}, local_count);
}

TEST_F(BitMaskTest, AllValids)
{
  const int num_rows = 100;

  bit_mask_t *bits = nullptr;
  EXPECT_EQ(GDF_SUCCESS, bit_mask::create_bit_mask(&bits, num_rows, 1));

  BitMask bit_mask(bits, num_rows);

  gdf_size_type local_count = 0;
  EXPECT_EQ(GDF_SUCCESS, count_bits(&local_count, bit_mask));

  EXPECT_EQ(GDF_SUCCESS, bit_mask::destroy_bit_mask(bits));

  EXPECT_EQ(gdf_size_type{100}, local_count);
}

TEST_F(BitMaskTest, FirstRowValid)
{
  const int num_rows = 4;

  bit_mask_t *bits = nullptr;
  EXPECT_EQ(GDF_SUCCESS, bit_mask::create_bit_mask(&bits, num_rows, 0));

  BitMask bit_mask(bits, num_rows);

  set_bit<<<1,1>>>(0, bit_mask);

  gdf_size_type local_count = 0;
  EXPECT_EQ(GDF_SUCCESS, count_bits(&local_count, bit_mask));

  EXPECT_EQ(gdf_size_type{1}, local_count);

  bit_mask_t temp = 0;
  bit_mask.get_element_host(0, temp);

  EXPECT_EQ(GDF_SUCCESS, bit_mask::destroy_bit_mask(bits));

  EXPECT_EQ(temp, bit_mask_t{0x1});
}

TEST_F(BitMaskTest, EveryOtherBit)
{
  const int num_rows = 8;

  bit_mask_t *bits = nullptr;
  EXPECT_EQ(GDF_SUCCESS, bit_mask::create_bit_mask(&bits, num_rows, 0));

  BitMask bit_mask(bits, num_rows);

  set_bit<<<1,1>>>(0, bit_mask);
  set_bit<<<1,1>>>(2, bit_mask);
  set_bit<<<1,1>>>(4, bit_mask);
  set_bit<<<1,1>>>(6, bit_mask);

  gdf_size_type local_count = 0;
  EXPECT_EQ(GDF_SUCCESS, count_bits(&local_count, bit_mask));

  EXPECT_EQ(gdf_size_type{4}, local_count);

  bit_mask_t temp = 0;
  bit_mask.get_element_host(0, temp);

  EXPECT_EQ(GDF_SUCCESS, bit_mask::destroy_bit_mask(bits));

  EXPECT_EQ(temp, bit_mask_t{0x55});
}

TEST_F(BitMaskTest, OtherEveryOtherBit)
{
  const int num_rows = 8;

  bit_mask_t *bits = nullptr;
  EXPECT_EQ(GDF_SUCCESS, bit_mask::create_bit_mask(&bits, num_rows, 0));

  BitMask bit_mask(bits, num_rows);

  set_bit<<<1,1>>>(1, bit_mask);
  set_bit<<<1,1>>>(3, bit_mask);
  set_bit<<<1,1>>>(5, bit_mask);
  set_bit<<<1,1>>>(7, bit_mask);

  gdf_size_type local_count = 0;
  EXPECT_EQ(GDF_SUCCESS, count_bits(&local_count, bit_mask));

  EXPECT_EQ(gdf_size_type{4}, local_count);

  bit_mask_t temp = 0;
  bit_mask.get_element_host(0, temp);

  EXPECT_EQ(GDF_SUCCESS, bit_mask::destroy_bit_mask(bits));

  EXPECT_EQ(temp, bit_mask_t{0xAA});
}

TEST_F(BitMaskTest, 15rows)
{
  const int num_rows = 15;

  bit_mask_t *bits = nullptr;
  EXPECT_EQ(GDF_SUCCESS, bit_mask::create_bit_mask(&bits, num_rows, 0));

  BitMask bit_mask(bits, num_rows);

  set_bit<<<1,1>>>(0, bit_mask);
  set_bit<<<1,1>>>(8, bit_mask);

  gdf_size_type local_count = 0;
  EXPECT_EQ(GDF_SUCCESS, count_bits(&local_count, bit_mask));

  EXPECT_EQ(gdf_size_type{2}, local_count);

  EXPECT_EQ(GDF_SUCCESS, bit_mask::destroy_bit_mask(bits));
}

TEST_F(BitMaskTest, 5rows)
{
  const int num_rows = 5;

  bit_mask_t *bits = nullptr;
  EXPECT_EQ(GDF_SUCCESS, bit_mask::create_bit_mask(&bits, num_rows, 0));

  BitMask bit_mask(bits, num_rows);

  set_bit<<<1,1>>>(0, bit_mask);

  gdf_size_type local_count = 0;
  EXPECT_EQ(GDF_SUCCESS, count_bits(&local_count, bit_mask));

  EXPECT_EQ(GDF_SUCCESS, bit_mask::destroy_bit_mask(bits));

  EXPECT_EQ(gdf_size_type{1}, local_count);
}

TEST_F(BitMaskTest, 10ValidRows)
{
  const int num_rows = 10;

  bit_mask_t *bits = nullptr;
  EXPECT_EQ(GDF_SUCCESS, bit_mask::create_bit_mask(&bits, num_rows, 1));

  BitMask bit_mask(bits, num_rows);

  gdf_size_type local_count = 0;
  EXPECT_EQ(GDF_SUCCESS, count_bits(&local_count, bit_mask));

  EXPECT_EQ(GDF_SUCCESS, bit_mask::destroy_bit_mask(bits));

  EXPECT_EQ(gdf_size_type{10}, local_count);
}

TEST_F(BitMaskTest, MultipleOfEight)
{
  const int num_rows = 1024;

  bit_mask_t *bits = nullptr;
  EXPECT_EQ(GDF_SUCCESS, bit_mask::create_bit_mask(&bits, num_rows, 0));

  BitMask bit_mask(bits, num_rows);

  for (int i = 0 ; i < num_rows ; i += 8) {
    set_bit<<<1,1>>>(i, bit_mask);
  }
  
  gdf_size_type local_count = 0;
  EXPECT_EQ(GDF_SUCCESS, count_bits(&local_count, bit_mask));

  EXPECT_EQ(GDF_SUCCESS, bit_mask::destroy_bit_mask(bits));

  EXPECT_EQ(gdf_size_type{128}, local_count);
}

TEST_F(BitMaskTest, NotMultipleOfEight)
{
  const int num_rows = 1023;

  bit_mask_t *bits = nullptr;
  EXPECT_EQ(GDF_SUCCESS, bit_mask::create_bit_mask(&bits, num_rows, 0));

  BitMask bit_mask(bits, num_rows);

  for (int i = 7 ; i < num_rows ; i += 8) {
    set_bit<<<1,1>>>(i, bit_mask);
  }
  
  gdf_size_type local_count = 0;
  EXPECT_EQ(GDF_SUCCESS, count_bits(&local_count, bit_mask));

  EXPECT_EQ(GDF_SUCCESS, bit_mask::destroy_bit_mask(bits));

  EXPECT_EQ(gdf_size_type{127}, local_count);
}

TEST_F(BitMaskTest, TenThousandRows)
{
  const int num_rows = 10000;

  bit_mask_t *bits = nullptr;
  EXPECT_EQ(GDF_SUCCESS, bit_mask::create_bit_mask(&bits, num_rows, 1));

  BitMask bit_mask(bits, num_rows);

  gdf_size_type local_count = 0;
  EXPECT_EQ(GDF_SUCCESS, count_bits(&local_count, bit_mask));

  EXPECT_EQ(GDF_SUCCESS, bit_mask::destroy_bit_mask(bits));

  EXPECT_EQ(gdf_size_type{10000}, local_count);
}

TEST_F(BitMaskTest, PerformanceTest)
{
  const int num_rows = 100000000;

  bit_mask_t *bits = nullptr;
  EXPECT_EQ(GDF_SUCCESS, bit_mask::create_bit_mask(&bits, num_rows, 0));

  BitMask bit_mask(bits, num_rows);

  int num_elements = bit_mask::num_elements(num_rows);
  int block_size = 256;
  int grid_size = (num_elements + block_size - 1)/block_size;

  uint32_t *local_valid = (uint32_t *) malloc(num_elements * sizeof(uint32_t));
  for (int i = 0 ; i < num_elements ; ++i) {
    local_valid[i] = 0x55555555U;
  }

  EXPECT_EQ(GDF_SUCCESS, bit_mask::copy_bit_mask(bit_mask.get_valid(), local_valid, num_rows, hipMemcpyHostToDevice));

  auto start = std::chrono::system_clock::now();
  hipProfilerStart();
  for(int i = 0; i < 1000; ++i) {
    gdf_size_type local_count = 0;
    count_bits(&local_count, bit_mask, grid_size, block_size);
  }
  hipProfilerStop();
  auto end = std::chrono::system_clock::now();
  std::chrono::duration<double> elapsed_seconds = end-start;
  std::cout << "Elapsed time (ms): " << elapsed_seconds.count()*1000 << std::endl;

  EXPECT_EQ(GDF_SUCCESS, bit_mask::destroy_bit_mask(bits));
  free(local_valid);
}

TEST_F(BitMaskTest, CudaThreadingTest)
{
  const int num_rows = 100000;
  bit_mask_t *bits = nullptr;
 
  EXPECT_EQ(GDF_SUCCESS, bit_mask::create_bit_mask(&bits, num_rows, 0));

  BitMask bit_mask(bits, num_rows);

  test_safe_set_clear_g<<<1,bit_mask::bits_per_element>>>(bit_mask);

  gdf_size_type local_count = 0;
  EXPECT_EQ(GDF_SUCCESS, count_bits(&local_count, bit_mask));

  EXPECT_EQ((gdf_size_type) (num_rows/2), local_count);

  EXPECT_EQ(GDF_SUCCESS, bit_mask::destroy_bit_mask(bits));
}

TEST_F(BitMaskTest, PaddingTest)
{
  //
  //  Set the number of rows to 32, we'll try padding to 
  //  256 bytes.
  //
  const int num_rows = 32;
  const int padding_bytes = 256;
  bit_mask_t *bits = nullptr;
 
  EXPECT_EQ(GDF_SUCCESS, bit_mask::create_bit_mask(&bits, num_rows, 1, padding_bytes));

  BitMask bit_mask(bits, num_rows);

  gdf_size_type local_count = 0;
  EXPECT_EQ(GDF_SUCCESS, count_bits(&local_count, bit_mask));

  EXPECT_EQ((gdf_size_type) num_rows, local_count);

  //
  //  To test this, we should be able to access the last element
  //
  int last_element = (padding_bytes / sizeof(bit_mask_t)) - 1;
  
  bit_mask_t temp = bit_mask_t{0};
  bit_mask.get_element_host(last_element, temp);
  EXPECT_EQ(~bit_mask_t{0}, temp);

  EXPECT_EQ(GDF_SUCCESS, bit_mask::destroy_bit_mask(bits));
}
