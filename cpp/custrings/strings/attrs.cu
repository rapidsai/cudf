/*
* Copyright (c) 2018-2019, NVIDIA CORPORATION.  All rights reserved.
*
* Licensed under the Apache License, Version 2.0 (the "License");
* you may not use this file except in compliance with the License.
* You may obtain a copy of the License at
*
*     http://www.apache.org/licenses/LICENSE-2.0
*
* Unless required by applicable law or agreed to in writing, software
* distributed under the License is distributed on an "AS IS" BASIS,
* WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
* See the License for the specific language governing permissions and
* limitations under the License.
*/

#include <hip/hip_runtime.h>
#include <thrust/device_vector.h>
#include <thrust/for_each.h>
#include <thrust/count.h>
#include <rmm/rmm.h>
#include <rmm/thrust_rmm_allocator.h>

#include "nvstrings/NVStrings.h"

#include "./NVStringsImpl.h"
#include "../custring_view.cuh"
#include "../unicode/is_flags.h"
#include "../util.h"

// this will return the number of characters for each string
unsigned int NVStrings::len(int* lengths, bool todevice)
{
    unsigned int count = size();
    if( lengths==0 || count==0 )
        return count;

    auto execpol = rmm::exec_policy(0);
    int* d_rtn = lengths;
    if( !todevice )
        d_rtn = device_alloc<int>(count,0);

    custring_view** d_strings = pImpl->getStringsPtr();
    thrust::for_each_n(execpol->on(0), thrust::make_counting_iterator<unsigned int>(0), count,
        [d_strings, d_rtn] __device__(unsigned int idx){
            custring_view* dstr = d_strings[idx];
            if( dstr )
                d_rtn[idx] = dstr->chars_count();
            else
                d_rtn[idx] = -1;
        });
    //
    //printCudaError(hipDeviceSynchronize(),"nvs-len");
    size_t size = thrust::reduce(execpol->on(0), d_rtn, d_rtn+count, (size_t)0,
         []__device__(int lhs, int rhs) {
            if( lhs < 0 )
                lhs = 0;
            if( rhs < 0 )
                rhs = 0;
            return lhs + rhs;
         });

    if( !todevice )
    {   // copy result back to host
        CUDA_TRY( hipMemcpyAsync(lengths,d_rtn,sizeof(int)*count,hipMemcpyDeviceToHost))
        RMM_FREE(d_rtn,0);
    }
    return (unsigned int)size;
}

// this will return the number of bytes for each string
size_t NVStrings::byte_count(int* lengths, bool todevice)
{
    unsigned int count = size();
    if( count==0 )
        return 0;

    auto execpol = rmm::exec_policy(0);
    int* d_rtn = lengths;
    if( !lengths )
        todevice = false; // makes sure we free correctly
    if( !todevice )
        d_rtn = device_alloc<int>(count,0);

    custring_view** d_strings = pImpl->getStringsPtr();
    thrust::for_each_n(execpol->on(0), thrust::make_counting_iterator<unsigned int>(0), count,
        [d_strings, d_rtn] __device__(unsigned int idx){
            custring_view* dstr = d_strings[idx];
            if( dstr )
                d_rtn[idx] = dstr->size();
            else
                d_rtn[idx] = -1;
        });
    //
    //printCudaError(hipDeviceSynchronize(),"nvs-bytes");
    size_t size = thrust::reduce(execpol->on(0), d_rtn, d_rtn+count, (size_t)0,
         []__device__(int lhs, int rhs) {
            if( lhs < 0 )
                lhs = 0;
            if( rhs < 0 )
                rhs = 0;
            return lhs + rhs;
         });
    if( !todevice )
    {   // copy result back to host
        if( lengths )
            CUDA_TRY( hipMemcpyAsync(lengths,d_rtn,sizeof(int)*count,hipMemcpyDeviceToHost))
        RMM_FREE(d_rtn,0);
    }
    return (unsigned int)size;
}


//
unsigned int NVStrings::isalnum( bool* results, bool todevice )
{
    unsigned int count = size();
    if( count==0 || results==0 )
        return 0;
    auto execpol = rmm::exec_policy(0);
    unsigned char* d_flags = get_unicode_flags();
    bool* d_rtn = results;
    if( !todevice )
        d_rtn = device_alloc<bool>(count,0);
    custring_view_array d_strings = pImpl->getStringsPtr();
    thrust::for_each_n(execpol->on(0), thrust::make_counting_iterator<unsigned int>(0), count,
        [d_strings, d_flags, d_rtn] __device__(unsigned int idx){
            custring_view* dstr = d_strings[idx];
            bool brc = false;
            if( dstr )
            {
                brc = !dstr->empty(); // alnum requires at least one character
                for( auto itr = dstr->begin(); brc && (itr != dstr->end()); itr++ )
                {
                    unsigned int uni = u82u(*itr);
                    unsigned int flg = uni <= 0x00FFFF ? d_flags[uni] : 0;
                    brc = IS_ALPHANUM(flg);
                }
            }
            d_rtn[idx] = brc;
        });
    // count the number of trues
    int matches = thrust::count(execpol->on(0), d_rtn, d_rtn+count, true );
    if( !todevice )
    {   // copy result back to host
        CUDA_TRY( hipMemcpyAsync(results,d_rtn,sizeof(bool)*count,hipMemcpyDeviceToHost))
        RMM_FREE(d_rtn,0);
    }
    return (unsigned int)matches;
}

unsigned int NVStrings::isalpha( bool* results, bool todevice )
{
    unsigned int count = size();
    if( count==0 || results==0 )
        return 0;
    auto execpol = rmm::exec_policy(0);
    unsigned char* d_flags = get_unicode_flags();
    bool* d_rtn = results;
    if( !todevice )
        d_rtn = device_alloc<bool>(count,0);
    custring_view_array d_strings = pImpl->getStringsPtr();
    thrust::for_each_n(execpol->on(0), thrust::make_counting_iterator<unsigned int>(0), count,
        [d_strings, d_flags, d_rtn] __device__(unsigned int idx){
            custring_view* dstr = d_strings[idx];
            bool brc = false;
            if( dstr )
            {
                brc = !dstr->empty(); // alpha requires at least one character
                for( auto itr = dstr->begin(); brc && (itr != dstr->end()); itr++ )
                {
                    unsigned int uni = u82u(*itr);
                    unsigned int flg = uni <= 0x00FFFF ? d_flags[uni] : 0;
                    brc = IS_ALPHA(flg);
                }
            }
            d_rtn[idx] = brc;
        });
    // count the number of trues
    int matches = thrust::count(execpol->on(0), d_rtn, d_rtn+count, true);
    if( !todevice )
    {   // copy result back to host
        CUDA_TRY( hipMemcpyAsync(results,d_rtn,sizeof(bool)*count,hipMemcpyDeviceToHost))
        RMM_FREE(d_rtn,0);
    }
    return (unsigned int)matches;
}

//
unsigned int NVStrings::isdigit( bool* results, bool todevice )
{
    unsigned int count = size();
    if( count==0 || results==0 )
        return 0;
    auto execpol = rmm::exec_policy(0);
    unsigned char* d_flags = get_unicode_flags();
    bool* d_rtn = results;
    if( !todevice )
        d_rtn = device_alloc<bool>(count,0);
    custring_view_array d_strings = pImpl->getStringsPtr();
    thrust::for_each_n(execpol->on(0), thrust::make_counting_iterator<unsigned int>(0), count,
        [d_strings, d_flags, d_rtn] __device__(unsigned int idx){
            custring_view* dstr = d_strings[idx];
            bool brc = false;
            if( dstr )
            {
                brc = !dstr->empty(); // digit requires at least one character
                for( auto itr = dstr->begin(); brc && (itr != dstr->end()); itr++ )
                {
                    unsigned int uni = u82u(*itr);
                    unsigned int flg = uni <= 0x00FFFF ? d_flags[uni] : 0;
                    brc = IS_DIGIT(flg);
                }
            }
            d_rtn[idx] = brc;
        });
    // count the number of trues
    int matches = thrust::count(execpol->on(0), d_rtn, d_rtn+count, true);
    if( !todevice )
    {   // copy result back to host
        CUDA_TRY( hipMemcpyAsync(results,d_rtn,sizeof(bool)*count,hipMemcpyDeviceToHost))
        RMM_FREE(d_rtn,0);
    }
    return (unsigned int)matches;
}

unsigned int NVStrings::isspace( bool* results, bool todevice )
{
    unsigned int count = size();
    if( count==0 || results==0 )
        return 0;
    auto execpol = rmm::exec_policy(0);
    unsigned char* d_flags = get_unicode_flags();
    bool* d_rtn = results;
    if( !todevice )
        d_rtn = device_alloc<bool>(count,0);
    custring_view_array d_strings = pImpl->getStringsPtr();
    thrust::for_each_n(execpol->on(0), thrust::make_counting_iterator<unsigned int>(0), count,
        [d_strings, d_flags, d_rtn] __device__(unsigned int idx){
            custring_view* dstr = d_strings[idx];
            bool brc = false;
            if( dstr )
            {
                brc = !dstr->empty(); // space requires at least one character
                for( auto itr = dstr->begin(); brc && (itr != dstr->end()); itr++ )
                {
                    unsigned int uni = u82u(*itr);
                    unsigned int flg = uni <= 0x00FFFF ? d_flags[uni] : 0;
                    brc = IS_SPACE(flg);
                }
            }
            d_rtn[idx] = brc;
        });
    // count the number of trues
    int matches = thrust::count(execpol->on(0), d_rtn, d_rtn+count, true);
    if( !todevice )
    {   // copy result back to host
        CUDA_TRY( hipMemcpyAsync(results,d_rtn,sizeof(bool)*count,hipMemcpyDeviceToHost))
        RMM_FREE(d_rtn,0);
    }
    return (unsigned int)matches;
}

unsigned int NVStrings::isdecimal( bool* results, bool todevice )
{
    unsigned int count = size();
    if( count==0 || results==0 )
        return 0;
    auto execpol = rmm::exec_policy(0);
    unsigned char* d_flags = get_unicode_flags();
    bool* d_rtn = results;
    if( !todevice )
        d_rtn = device_alloc<bool>(count,0);
    custring_view_array d_strings = pImpl->getStringsPtr();
    thrust::for_each_n(execpol->on(0), thrust::make_counting_iterator<unsigned int>(0), count,
        [d_strings, d_flags, d_rtn] __device__(unsigned int idx){
            custring_view* dstr = d_strings[idx];
            bool brc = false;
            if( dstr )
            {
                brc = !dstr->empty(); // decimal requires at least one character
                for( auto itr = dstr->begin(); brc && (itr != dstr->end()); itr++ )
                {
                    unsigned int uni = u82u(*itr);
                    unsigned int flg = uni <= 0x00FFFF ? d_flags[uni] : 0;
                    brc = IS_DECIMAL(flg);
                }
            }
            d_rtn[idx] = brc;
        });
    // count the number of trues
    int matches = thrust::count(execpol->on(0), d_rtn, d_rtn+count, true);
    if( !todevice )
    {   // copy result back to host
        CUDA_TRY( hipMemcpyAsync(results,d_rtn,sizeof(bool)*count,hipMemcpyDeviceToHost))
        RMM_FREE(d_rtn,0);
    }
    return (unsigned int)matches;
}

unsigned int NVStrings::isnumeric( bool* results, bool todevice )
{
    unsigned int count = size();
    if( count==0 || results==0 )
        return 0;
    auto execpol = rmm::exec_policy(0);
    unsigned char* d_flags = get_unicode_flags();
    bool* d_rtn = results;
    if( !todevice )
        d_rtn = device_alloc<bool>(count,0);
    custring_view_array d_strings = pImpl->getStringsPtr();
    thrust::for_each_n(execpol->on(0), thrust::make_counting_iterator<unsigned int>(0), count,
        [d_strings, d_flags, d_rtn] __device__(unsigned int idx){
            custring_view* dstr = d_strings[idx];
            bool brc = false;
            if( dstr )
            {
                brc = !dstr->empty(); // numeric requires at least one character
                for( auto itr = dstr->begin(); brc && (itr != dstr->end()); itr++ )
                {
                    unsigned int uni = u82u(*itr);
                    unsigned int flg = uni <= 0x00FFFF ? d_flags[uni] : 0;
                    brc = IS_NUMERIC(flg);
                }
            }
            d_rtn[idx] = brc;
        });
    // count the number of trues
    int matches = thrust::count(execpol->on(0), d_rtn, d_rtn+count, true);
    if( !todevice )
    {   // copy result back to host
        CUDA_TRY( hipMemcpyAsync(results,d_rtn,sizeof(bool)*count,hipMemcpyDeviceToHost))
        RMM_FREE(d_rtn,0);
    }
    return (unsigned int)matches;
}

unsigned int NVStrings::islower( bool* results, bool todevice )
{
    unsigned int count = size();
    if( count==0 || results==0 )
        return 0;
    auto execpol = rmm::exec_policy(0);
    unsigned char* d_flags = get_unicode_flags();
    bool* d_rtn = results;
    if( !todevice )
        d_rtn = device_alloc<bool>(count,0);
    custring_view_array d_strings = pImpl->getStringsPtr();
    thrust::for_each_n(execpol->on(0), thrust::make_counting_iterator<unsigned int>(0), count,
        [d_strings, d_flags, d_rtn] __device__(unsigned int idx){
            custring_view* dstr = d_strings[idx];
            bool brc = false;
            if( dstr )
            {
                brc = !dstr->empty(); // requires at least one character
                for( auto itr = dstr->begin(); brc && (itr != dstr->end()); itr++ )
                {
                    unsigned int uni = u82u(*itr);
                    unsigned int flg = (uni <= 0x00FFFF ? d_flags[uni] : 0);
                    brc = !IS_ALPHA(flg) || IS_LOWER(flg);
                }
            }
            d_rtn[idx] = brc;
        });
    // count the number of trues
    int matches = thrust::count(execpol->on(0), d_rtn, d_rtn+count, true);
    if( !todevice )
    {   // copy result back to host
        CUDA_TRY( hipMemcpyAsync(results,d_rtn,sizeof(bool)*count,hipMemcpyDeviceToHost))
        RMM_FREE(d_rtn,0);
    }
    return (unsigned int)matches;
}

unsigned int NVStrings::isupper( bool* results, bool todevice )
{
    unsigned int count = size();
    if( count==0 || results==0 )
        return 0;
    auto execpol = rmm::exec_policy(0);
    unsigned char* d_flags = get_unicode_flags();
    bool* d_rtn = results;
    if( !todevice )
        d_rtn = device_alloc<bool>(count,0);
    custring_view_array d_strings = pImpl->getStringsPtr();
    thrust::for_each_n(execpol->on(0), thrust::make_counting_iterator<unsigned int>(0), count,
        [d_strings, d_flags, d_rtn] __device__(unsigned int idx){
            custring_view* dstr = d_strings[idx];
            bool brc = false;
            if( dstr )
            {
                brc = !dstr->empty(); // requires at least one character
                for( auto itr = dstr->begin(); brc && (itr != dstr->end()); itr++ )
                {
                    unsigned int uni = u82u(*itr);
                    unsigned int flg = (uni <= 0x00FFFF ? d_flags[uni] : 0);
                    brc = !IS_ALPHA(flg) || IS_UPPER(flg);
                }
            }
            d_rtn[idx] = brc;
        });
    // count the number of trues
    int matches = thrust::count(execpol->on(0), d_rtn, d_rtn+count, true);
    if( !todevice )
    {   // copy result back to host
        CUDA_TRY( hipMemcpyAsync(results,d_rtn,sizeof(bool)*count,hipMemcpyDeviceToHost))
        RMM_FREE(d_rtn,0);
    }
    return (unsigned int)matches;
}

unsigned int NVStrings::is_empty( bool* results, bool todevice )
{
    unsigned int count = size();
    if( count==0 || results==0 )
        return 0;
    auto execpol = rmm::exec_policy(0);
    bool* d_rtn = results;
    if( !todevice )
        d_rtn = device_alloc<bool>(count,0);
    custring_view_array d_strings = pImpl->getStringsPtr();
    thrust::for_each_n(execpol->on(0), thrust::make_counting_iterator<unsigned int>(0), count,
        [d_strings, d_rtn] __device__(unsigned int idx){
            custring_view* dstr = d_strings[idx];
            bool brc = true; // null is empty
            if( dstr )
                brc = dstr->empty(); // requires at least one character
            d_rtn[idx] = brc;
        });
    // count the number of trues
    int matches = thrust::count(execpol->on(0), d_rtn, d_rtn+count, true);
    if( !todevice )
    {   // copy result back to host
        CUDA_TRY( hipMemcpyAsync(results,d_rtn,sizeof(bool)*count,hipMemcpyDeviceToHost))
        RMM_FREE(d_rtn,0);
    }
    return (unsigned int)matches;
}

