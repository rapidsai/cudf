#include "hip/hip_runtime.h"
/*
* Copyright (c) 2018-2019, NVIDIA CORPORATION.  All rights reserved.
*
* Licensed under the Apache License, Version 2.0 (the "License");
* you may not use this file except in compliance with the License.
* You may obtain a copy of the License at
*
*     http://www.apache.org/licenses/LICENSE-2.0
*
* Unless required by applicable law or agreed to in writing, software
* distributed under the License is distributed on an "AS IS" BASIS,
* WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
* See the License for the specific language governing permissions and
* limitations under the License.
*/

#include <cstdio>
#include <exception>
#include <sstream>
#include <hip/hip_runtime.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/for_each.h>
#include <thrust/sort.h>
#include <rmm/rmm.h>
#include <rmm/thrust_rmm_allocator.h>
#include <utilities/error_utils.hpp>

#include "nvstrings/NVStrings.h"

#include "./NVStringsImpl.h"
#include "../custring_view.cuh"
#include "../util.h"

//
NVStrings* NVStrings::slice_replace( const char* repl, int start, int stop )
{
    if( !repl )
        throw std::invalid_argument("nvstrings::slice_replace parameter cannot be null");
    auto execpol = rmm::exec_policy(0);
    unsigned int replen = (unsigned int)strlen(repl);
    char* d_repl = device_alloc<char>(replen,0);
    CUDA_TRY( hipMemcpyAsync(d_repl,repl,replen,hipMemcpyHostToDevice))
    // compute size of output buffer
    unsigned int count = size();
    custring_view_array d_strings = pImpl->getStringsPtr();
    rmm::device_vector<size_t> lengths(count,0);
    size_t* d_lengths = lengths.data().get();
    thrust::for_each_n(execpol->on(0), thrust::make_counting_iterator<unsigned int>(0), count,
        [d_strings, d_repl, replen, start, stop, d_lengths] __device__(unsigned int idx){
            custring_view* dstr = d_strings[idx];
            if( !dstr )
                return;
            unsigned int len = 0;
            if( start < dstr->chars_count() )
                len = dstr->replace_size((unsigned)start,(unsigned)(stop-start),d_repl,replen);
            else
            {   // another odd pandas case: if out-of-bounds, just append
                int bytes = dstr->size() + replen;
                int nchars = dstr->chars_count() + custring_view::chars_in_string(d_repl,replen);
                len = custring_view::alloc_size(bytes,nchars);
            }
            len = ALIGN_SIZE(len);
            d_lengths[idx] = (size_t)len;
        });

    // create output object
    NVStrings* rtn = new NVStrings(count);
    char* d_buffer = rtn->pImpl->createMemoryFor(d_lengths);
    if( d_buffer==0 )
    {
        if( d_repl )
            RMM_FREE(d_repl,0);
        return rtn;
    }
    // create offsets
    rmm::device_vector<size_t> offsets(count,0);
    thrust::exclusive_scan(execpol->on(0),lengths.begin(),lengths.end(),offsets.begin());
    // do the slice and replace
    custring_view_array d_results = rtn->pImpl->getStringsPtr();
    size_t* d_offsets = offsets.data().get();
    thrust::for_each_n(execpol->on(0), thrust::make_counting_iterator<size_t>(0), count,
        [d_strings, d_repl, replen, start, stop, d_buffer, d_offsets, d_results] __device__(size_t idx){
            custring_view* dstr = d_strings[idx];
            if( !dstr )
                return;
            char* buffer = d_buffer + d_offsets[idx];
            custring_view* dout = nullptr;
            if( start < dstr->chars_count() )
                dout = dstr->replace((unsigned)start,(unsigned)(stop-start),d_repl,replen,buffer);
            else
            {   // append for pandas consistency
                int bytes = dstr->size();
                char* ptr = buffer;
                memcpy( ptr, dstr->data(), bytes );
                ptr += bytes;
                memcpy( ptr, d_repl, replen );
                bytes += replen;
                dout = custring_view::create_from(buffer,buffer,bytes);
            }
            d_results[idx] = dout;
        });
    //
    if( d_repl )
        RMM_FREE(d_repl,0);
    return rtn;
}

// this should replace multiple occurrences up to maxrepl
NVStrings* NVStrings::replace( const char* str, const char* repl, int maxrepl )
{
    if( !str || !*str )
        throw std::invalid_argument("replace parameter cannot be null or empty");
    auto execpol = rmm::exec_policy(0);
    custring_view* d_str = custring_from_host(str);
    if( !repl )
        repl = "";
    custring_view* d_repl = custring_from_host(repl);

    // compute size of the output
    unsigned int count = size();
    custring_view** d_strings = pImpl->getStringsPtr();
    rmm::device_vector<size_t> sizes(count,0);
    size_t* d_sizes = sizes.data().get();
    thrust::for_each_n(execpol->on(0), thrust::make_counting_iterator<unsigned int>(0), count,
        [d_strings, d_str, d_repl, maxrepl, d_sizes] __device__(unsigned int idx){
            custring_view* dstr = d_strings[idx];
            if( !dstr )
                return;
            int mxn = maxrepl;
            if( mxn < 0 )
                mxn = dstr->chars_count(); //max possible replaces for this string
            unsigned int bytes = dstr->size(), nchars = dstr->chars_count();
            int pos = dstr->find(*d_str);
            // counting bytes and chars
            while((pos >= 0) && (mxn > 0))
            {
                bytes += d_repl->size() - d_str->size();
                nchars += d_repl->chars_count() - d_str->chars_count();
                pos = dstr->find(*d_str,(unsigned)pos+d_str->chars_count()); // next one
                --mxn;
            }
            unsigned int size = custring_view::alloc_size(bytes,nchars);
            d_sizes[idx] = ALIGN_SIZE(size);
        });
    //
    // create output object
    NVStrings* rtn = new NVStrings(count);
    char* d_buffer = rtn->pImpl->createMemoryFor(d_sizes);
    if( d_buffer==0 )
    {
        RMM_FREE(d_str,0);
        RMM_FREE(d_repl,0);
        return rtn; // all strings are null
    }
    // create offsets
    rmm::device_vector<size_t> offsets(count,0);
    thrust::exclusive_scan(execpol->on(0),sizes.begin(),sizes.end(),offsets.begin());
    // do the thing
    custring_view_array d_results = rtn->pImpl->getStringsPtr();
    size_t* d_offsets = offsets.data().get();
    thrust::for_each_n(execpol->on(0), thrust::make_counting_iterator<unsigned int>(0), count,
        [d_strings, d_str, d_repl, d_buffer, d_offsets, maxrepl, d_results] __device__(unsigned int idx){
            custring_view* dstr = d_strings[idx];
            if( !dstr )
                return;
            int mxn = maxrepl;
            if( mxn < 0 )
                mxn = dstr->chars_count(); //max possible replaces for this string
            //
            char* buffer = d_buffer + d_offsets[idx];
            char* sptr = dstr->data();
            char* optr = buffer;
            unsigned int size = dstr->size();
            int pos = dstr->find(*d_str), lpos=0;
            while((pos >= 0) && (mxn > 0))
            {                                                      // i:bbbbsssseeee
                int spos = dstr->byte_offset_for(pos);             //       ^
                copy_and_incr(optr,sptr+lpos,spos-lpos);           // o:bbbb
                copy_and_incr(optr,d_repl->data(),d_repl->size()); // o:bbbbrrrr
                lpos = spos + d_str->size();//ssz;                 // i:bbbbsssseeee
                pos = dstr->find(*d_str,pos+d_str->chars_count()); //           ^
                --mxn;
            }
            memcpy(optr,sptr+lpos,size-lpos);                      // o:bbbbrrrreeee
            unsigned int nsz = (unsigned int)(optr - buffer) + size - lpos;
            d_results[idx] = custring_view::create_from(buffer,buffer,nsz);
        });
    //
    RMM_FREE(d_str,0);
    RMM_FREE(d_repl,0);
    return rtn;
}

// used by both mult-replaces below
// also does the size calculations inline
struct replace_multi_fn
{
    custring_view_array d_strings;
    custring_view_array d_targets;
    unsigned int target_count;
    custring_view_array d_repls;
    unsigned int repl_count;
    size_t* d_offsets;
    bool bcompute_size_only{true};
    char* d_buffer;
    custring_view_array d_results;

    __device__ void operator()(unsigned int idx)
    {
        custring_view* dstr = d_strings[idx];
        if( !dstr )
            return;
        char* buffer = nullptr;
        if( !bcompute_size_only )
            buffer = d_buffer + d_offsets[idx];
        char* optr = buffer;
        unsigned int nbytes = dstr->size(), nchars = dstr->chars_count();
        char* sptr = dstr->data();
        unsigned int size = nbytes, spos = 0, lpos = 0;
        while( spos < size )
        {   // check each character against each target
            for( int tidx=0; tidx < target_count; ++tidx )
            {
                custring_view* dtgt = d_targets[tidx];
                if( dtgt && // skip over any nulls
                    (dtgt->size() <= (size-spos)) && // check fit
                    (dtgt->compare(sptr+spos,dtgt->size())==0) ) // does it match
                {   // found one
                    custring_view* d_repl = (repl_count==1 ? d_repls[0]:d_repls[tidx]);
                    if( bcompute_size_only )
                    {
                        nbytes += (d_repl ? d_repl->size():0) - dtgt->size();
                        nchars += (d_repl ? d_repl->chars_count():0) - dtgt->chars_count();
                    }
                    else
                    {
                        copy_and_incr(optr,sptr+lpos,spos-lpos);               // copy left
                        if( d_repl )                                           // and
                            copy_and_incr(optr,d_repl->data(),d_repl->size()); // replace
                        lpos = spos + dtgt->size();
                    }
                    spos += dtgt->size()-1;
                    break;
                }
            }
            ++spos;
        }
        if( bcompute_size_only )
        {
            unsigned int nsize = custring_view::alloc_size(nbytes,nchars);
            d_offsets[idx] = ALIGN_SIZE(nsize);
        }
        else
        {
            memcpy(optr,sptr+lpos,size-lpos); // copy remainder
            unsigned int nsz = (unsigned int)(optr - buffer) + size - lpos;
            d_results[idx] = custring_view::create_from(buffer,buffer,nsz);
        }
    }
};

//
NVStrings* NVStrings::replace( NVStrings& targets, NVStrings& repls )
{
    if( targets.size()==0 || repls.size()==0 )
        throw std::invalid_argument("replace targets and repls parameters cannot be empty");
    if( repls.size()>1 && (repls.size() != targets.size()) )
        throw std::invalid_argument("replace targets and replacement sizes must match");
    auto execpol = rmm::exec_policy(0);

    // compute size of the output
    custring_view** d_strings = pImpl->getStringsPtr();
    unsigned int count = size();
    custring_view** d_targets = targets.pImpl->getStringsPtr();
    unsigned int target_count = targets.size();
    custring_view_array d_repls = repls.pImpl->getStringsPtr();
    unsigned int repl_count = repls.size();
    rmm::device_vector<size_t> sizes(count,0);
    size_t* d_sizes = sizes.data().get();
    // get the sizes
    thrust::for_each_n(execpol->on(0), thrust::make_counting_iterator<unsigned int>(0), count,
        replace_multi_fn{d_strings, d_targets, target_count, d_repls, repl_count, d_sizes} );
    //
    // create output object
    NVStrings* rtn = new NVStrings(count);
    char* d_buffer = rtn->pImpl->createMemoryFor(d_sizes);
    if( d_buffer==0 )
        return rtn; // all strings are null
    // create offsets
    rmm::device_vector<size_t> offsets(count,0);
    thrust::exclusive_scan(execpol->on(0),sizes.begin(),sizes.end(),offsets.begin());
    // do the thing
    custring_view_array d_results = rtn->pImpl->getStringsPtr();
    size_t* d_offsets = offsets.data().get();
    thrust::for_each_n(execpol->on(0), thrust::make_counting_iterator<unsigned int>(0), count,
        replace_multi_fn{d_strings, d_targets, target_count, d_repls, repl_count, d_offsets, false, d_buffer, d_results });
    //
    return rtn;
}

//
NVStrings* NVStrings::translate( std::pair<unsigned,unsigned>* utable, unsigned int tableSize )
{
    unsigned int count = size();
    custring_view_array d_strings = pImpl->getStringsPtr();
    auto execpol = rmm::exec_policy(0);

    // convert unicode table into utf8 table
    thrust::host_vector< thrust::pair<Char,Char> > htable(tableSize);
    for( unsigned int idx=0; idx < tableSize; ++idx )
    {
        htable[idx].first = u2u8(utable[idx].first);
        htable[idx].second = u2u8(utable[idx].second);
    }
    // could sort on the device; this table should not be very big
    thrust::sort(thrust::host, htable.begin(), htable.end(),
        [] __host__ (thrust::pair<Char,Char> p1, thrust::pair<Char,Char> p2) { return p1.first > p2.first; });

    // copy translate table to device memory
    rmm::device_vector< thrust::pair<Char,Char> > table(htable);
    thrust::pair<Char,Char>* d_table = table.data().get();

    // compute size of each new string
    rmm::device_vector<size_t> sizes(count,0);
    size_t* d_sizes = sizes.data().get();
    int tsize = tableSize;
    thrust::for_each_n(execpol->on(0), thrust::make_counting_iterator<unsigned int>(0), count,
        [d_strings, d_table, tsize, d_sizes] __device__(unsigned int idx){
            custring_view* dstr = d_strings[idx];
            if( !dstr )
                return;
            const char* sptr = dstr->data();
            unsigned int bytes = dstr->size();
            unsigned int nchars = dstr->chars_count();
            for( unsigned int i=0; i < nchars; ++i )
            {
                Char ch = dstr->at(i);
                Char nch = ch;
                for( int t=0; t < tsize; ++t ) // replace with faster lookup
                    nch = ( ch==d_table[t].first ? d_table[t].second : nch );
                int bic = custring_view::bytes_in_char(ch);
                int nbic = (nch ? custring_view::bytes_in_char(nch) : 0);
                bytes += nbic - bic;
                if( nch==0 )
                    --nchars;
            }
            unsigned int size = custring_view::alloc_size(bytes,nchars);
            d_sizes[idx] = ALIGN_SIZE(size);
        });

    // create output object
    NVStrings* rtn = new NVStrings(count);
    char* d_buffer = rtn->pImpl->createMemoryFor(d_sizes);
    if( d_buffer==0 )
        return rtn;
    // create offsets
    rmm::device_vector<size_t> offsets(count,0);
    thrust::exclusive_scan(execpol->on(0),sizes.begin(),sizes.end(),offsets.begin());
    // do the translate
    custring_view_array d_results = rtn->pImpl->getStringsPtr();
    size_t* d_offsets = offsets.data().get();
    thrust::for_each_n(execpol->on(0), thrust::make_counting_iterator<unsigned int>(0), count,
        [d_strings, d_buffer, d_offsets, d_table, tsize, d_results] __device__(unsigned int idx){
            custring_view* dstr = d_strings[idx];
            if( !dstr )
                return;
            char* buffer = d_buffer + d_offsets[idx];
            const char* sptr = dstr->data();
            unsigned int nchars = dstr->chars_count();
            char* optr = buffer;
            int nsz = 0;
            for( unsigned int i=0; i < nchars; ++i )
            {
                Char ch = 0;
                unsigned int cw = custring_view::char_to_Char(sptr,ch);
                Char nch = ch;
                for( int t=0; t < tsize; ++t ) // replace with faster lookup
                    nch = ( ch==d_table[t].first ? d_table[t].second : nch );
                sptr += cw;
                if( nch==0 )
                    continue;
                unsigned int nbic = custring_view::Char_to_char(nch,optr);
                optr += nbic;
                nsz += nbic;
            }
            d_results[idx] = custring_view::create_from(buffer,buffer,nsz);
        });
    //
    return rtn;
}

//
// This will create a new instance replacing any nulls with the provided string.
// The parameter can be an empty string or any other string but not null.
NVStrings* NVStrings::fillna( const char* str )
{
    if( str==0 )
        throw std::invalid_argument("nvstrings::fillna parameter cannot be null");
    auto execpol = rmm::exec_policy(0);
    unsigned int ssz = (unsigned int)strlen(str);
    unsigned int asz = custring_view::alloc_size(str,ssz);
    char* d_str = device_alloc<char>(ssz+1,0);
    CUDA_TRY( hipMemcpyAsync(d_str,str,ssz+1,hipMemcpyHostToDevice))

    // compute size of the output
    unsigned int count = size();
    custring_view** d_strings = pImpl->getStringsPtr();
    rmm::device_vector<size_t> sizes(count,0);
    size_t* d_sizes = sizes.data().get();
    thrust::for_each_n(execpol->on(0), thrust::make_counting_iterator<unsigned int>(0), count,
        [d_strings, d_str, asz, d_sizes] __device__(unsigned int idx){
            custring_view* dstr = d_strings[idx];
            unsigned int size = asz;
            if( dstr )
                size = dstr->alloc_size();
            d_sizes[idx] = ALIGN_SIZE(size);
        });
    //
    NVStrings* rtn = new NVStrings(count); // create output object
    char* d_buffer = rtn->pImpl->createMemoryFor(d_sizes);
    rmm::device_vector<size_t> offsets(count,0); // create offsets
    thrust::exclusive_scan(execpol->on(0),sizes.begin(),sizes.end(),offsets.begin());
    // do the thing
    custring_view_array d_results = rtn->pImpl->getStringsPtr();
    size_t* d_offsets = offsets.data().get();
    thrust::for_each_n(execpol->on(0), thrust::make_counting_iterator<unsigned int>(0), count,
        [d_strings, d_str, ssz, d_buffer, d_offsets, d_results] __device__(unsigned int idx){
            custring_view* dstr = d_strings[idx];
            char* buffer = d_buffer + d_offsets[idx];
            if( dstr )
                dstr = custring_view::create_from(buffer,*dstr);
            else
                dstr = custring_view::create_from(buffer,d_str,ssz);
            d_results[idx] = dstr;
        });
    //
    RMM_FREE(d_str,0);
    return rtn;
}


// This will create a new instance replacing any nulls with the provided strings.
// The strings are matched by index. Non-null strings are not replaced.
NVStrings* NVStrings::fillna( NVStrings& strs )
{
    if( strs.size()!=size() )
        throw std::invalid_argument("nvstrings::fillna parameter must have the same number of strings");
    auto execpol = rmm::exec_policy(0);

    // compute size of the output
    auto count = size();
    custring_view** d_strings = pImpl->getStringsPtr();
    custring_view** d_repls = strs.pImpl->getStringsPtr();
    rmm::device_vector<size_t> sizes(count,0);
    size_t* d_sizes = sizes.data().get();
    thrust::for_each_n(execpol->on(0), thrust::make_counting_iterator<unsigned int>(0), count,
        [d_strings, d_repls, d_sizes] __device__(unsigned int idx){
            custring_view* dstr = d_strings[idx];
            custring_view* drepl = d_repls[idx];
            unsigned int size = 0;
            if( dstr )
                size = dstr->alloc_size();
            else if( drepl )
                size = drepl->alloc_size();
            else
                return; // both are null
            d_sizes[idx] = ALIGN_SIZE(size);
        });
    //
    NVStrings* rtn = new NVStrings(count); // create output object
    char* d_buffer = rtn->pImpl->createMemoryFor(d_sizes);
    rmm::device_vector<size_t> offsets(count,0); // create offsets
    thrust::exclusive_scan(execpol->on(0),sizes.begin(),sizes.end(),offsets.begin());
    // do the thing
    custring_view_array d_results = rtn->pImpl->getStringsPtr();
    size_t* d_offsets = offsets.data().get();
    thrust::for_each_n(execpol->on(0), thrust::make_counting_iterator<unsigned int>(0), count,
        [d_strings, d_repls, d_buffer, d_offsets, d_results] __device__(unsigned int idx){
            custring_view* dstr = d_strings[idx];
            custring_view* drepl = d_repls[idx];
            char* buffer = d_buffer + d_offsets[idx];
            if( dstr )
                d_results[idx] = custring_view::create_from(buffer,*dstr);
            else if( drepl )
                d_results[idx] = custring_view::create_from(buffer,*drepl);
        });
    //
    return rtn;
}

//
// The slice_replace method can do this too.
// This is easier to use and more efficient.
NVStrings* NVStrings::insert( const char* repl, int start )
{
    if( !repl )
        throw std::invalid_argument("nvstrings::slice_replace parameter cannot be null");
    auto execpol = rmm::exec_policy(0);
    unsigned int replen = (unsigned int)strlen(repl);
    char* d_repl = device_alloc<char>(replen,0);
    CUDA_TRY( hipMemcpyAsync(d_repl,repl,replen,hipMemcpyHostToDevice))
    // compute size of output buffer
    unsigned int count = size();
    custring_view_array d_strings = pImpl->getStringsPtr();
    rmm::device_vector<size_t> lengths(count,0);
    size_t* d_lengths = lengths.data().get();
    thrust::for_each_n(execpol->on(0), thrust::make_counting_iterator<unsigned int>(0), count,
        [d_strings, d_repl, replen, start, d_lengths] __device__(unsigned int idx){
            custring_view* dstr = d_strings[idx];
            if( !dstr )
                return;
            unsigned int len = dstr->alloc_size();
            if( start <= (int)dstr->chars_count() )
                len = dstr->insert_size(d_repl,replen);
            len = ALIGN_SIZE(len);
            d_lengths[idx] = (size_t)len;
        });

    // create output object
    NVStrings* rtn = new NVStrings(count);
    char* d_buffer = rtn->pImpl->createMemoryFor(d_lengths);
    if( d_buffer==0 )
    {
        if( d_repl )
            RMM_FREE(d_repl,0);
        return rtn;
    }
    // create offsets
    rmm::device_vector<size_t> offsets(count,0);
    thrust::exclusive_scan(execpol->on(0),lengths.begin(),lengths.end(),offsets.begin());
    // do the insert
    custring_view_array d_results = rtn->pImpl->getStringsPtr();
    size_t* d_offsets = offsets.data().get();
    thrust::for_each_n(execpol->on(0), thrust::make_counting_iterator<size_t>(0), count,
        [d_strings, d_repl, replen, start, d_buffer, d_offsets, d_results] __device__(size_t idx){
            custring_view* dstr = d_strings[idx];
            if( !dstr )
                return;
            char* buffer = d_buffer + d_offsets[idx];
            custring_view* dout = custring_view::create_from(buffer,*dstr);
            if( start <= (int)dstr->chars_count() )
            {
                unsigned int pos = ( start < 0 ? dstr->chars_count() : (unsigned)start );
                dout->insert(pos,d_repl,replen);
            }
            d_results[idx] = dout;
        });
    //
    if( d_repl )
        RMM_FREE(d_repl,0);
    return rtn;
}
