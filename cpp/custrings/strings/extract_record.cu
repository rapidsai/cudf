/*
* Copyright (c) 2018-2019, NVIDIA CORPORATION.  All rights reserved.
*
* Licensed under the Apache License, Version 2.0 (the "License");
* you may not use this file except in compliance with the License.
* You may obtain a copy of the License at
*
*     http://www.apache.org/licenses/LICENSE-2.0
*
* Unless required by applicable law or agreed to in writing, software
* distributed under the License is distributed on an "AS IS" BASIS,
* WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
* See the License for the specific language governing permissions and
* limitations under the License.
*/

#include <exception>
#include <sstream>
#include <hip/hip_runtime.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/for_each.h>
#include <rmm/rmm.h>
#include <rmm/thrust_rmm_allocator.h>

#include "nvstrings/NVStrings.h"

#include "./NVStringsImpl.h"
#include "../custring_view.cuh"
#include "../regex/regex.cuh"
#include "../unicode/is_flags.h"
#include "../util.h"

//
// This functor is used with the extract_record method in two ways.
// First, it computes the output size of each string.
// The extract_record needs this in order allocate the memory required
// for the output -- new instance per string.
// The 2nd call does the actual extract into new memory provided.
// Combining the two into a single functor doubles the speed of the build
// since inlining of the regex code is minimized.
// There should be no divergence since all kernels pass through either
// the compute-size branch or not -- there is no mixture within the same
// kernel launch.
//
template<size_t stack_size>
struct extract_record_fn
{
    dreprog* prog;
    custring_view_array d_strings;
    int groups;
    int* d_lengths;
    bool bcompute_size_only{true};
    char** d_buffers;
    custring_view_array* d_rows;
    __device__ void operator()(unsigned int idx)
    {
        custring_view* dstr = d_strings[idx];
        if( !dstr )
            return;
        u_char data1[stack_size], data2[stack_size];
        prog->set_stack_mem(data1,data2);
        int begin = 0, end = dstr->chars_count();
        if( prog->find(idx,dstr,begin,end) <=0 )
            return;
        int* sizes = d_lengths + (idx*groups);
        char* buffer = nullptr;
        custring_view_array d_row = nullptr;
        if( !bcompute_size_only )
        {
            buffer = (char*)d_buffers[idx];
            d_row = d_rows[idx];
        }
        for( int col=0; col < groups; ++col )
        {
            int spos=begin, epos=end;
            if( prog->extract(idx,dstr,spos,epos,col) <=0 )
                continue;
            if( bcompute_size_only )
            {
                unsigned int size = dstr->substr_size(spos,epos-spos);
                sizes[col] = (size_t)ALIGN_SIZE(size);
            }
            else
            {
                d_row[col] = dstr->substr((unsigned)spos,(unsigned)(epos-spos),1,buffer);
                buffer += sizes[col];
            }
        }
    }
};

//
// Extract strings into new instance per string as specified and found by the given regex pattern.
//
int NVStrings::extract_record( const char* pattern, std::vector<NVStrings*>& results)
{
    if( pattern==0 )
        return -1;
    unsigned int count = size();
    if( count==0 )
        return 0;

    auto execpol = rmm::exec_policy(0);
    // compile regex into device object
    const char32_t* ptn32 = to_char32(pattern);
    dreprog* prog = dreprog::create_from(ptn32,get_unicode_flags());
    delete ptn32;
    // allocate regex working memory if necessary
    int regex_insts = prog->inst_counts();
    if( regex_insts > MAX_STACK_INSTS )
    {
        if( !prog->alloc_relists(count) )
        {
            std::ostringstream message;
            message << "nvstrings::extract_record: number of instructions (" << prog->inst_counts() << ") ";
            message << "and number of strings (" << count << ") ";
            message << "exceeds available memory";
            dreprog::destroy(prog);
            throw std::invalid_argument(message.str());
        }
    }
    //
    int groups = prog->group_counts();
    if( groups==0 )
    {
        dreprog::destroy(prog);
        return 0;
    }
    // compute lengths of each group for each string
    custring_view_array d_strings = pImpl->getStringsPtr();
    rmm::device_vector<int> lengths(count*groups,0);
    int* d_lengths = lengths.data().get();
    if( (regex_insts > MAX_STACK_INSTS) || (regex_insts <= 10) )
        thrust::for_each_n(execpol->on(0), thrust::make_counting_iterator<unsigned int>(0), count,
            extract_record_fn<RX_STACK_SMALL>{prog, d_strings, groups, d_lengths});
    else if( regex_insts <= 100 )
        thrust::for_each_n(execpol->on(0), thrust::make_counting_iterator<unsigned int>(0), count,
            extract_record_fn<RX_STACK_MEDIUM>{prog, d_strings, groups, d_lengths});
    else
        thrust::for_each_n(execpol->on(0), thrust::make_counting_iterator<unsigned int>(0), count,
            extract_record_fn<RX_STACK_LARGE>{prog, d_strings, groups, d_lengths});
    //
    hipDeviceSynchronize();
    // this part will be slow for large number of strings
    rmm::device_vector<custring_view_array> strings(count,nullptr);
    rmm::device_vector<char*> buffers(count,nullptr);
    for( unsigned int idx=0; idx < count; ++idx )
    {
        NVStrings* row = new NVStrings(groups);
        results.push_back(row);
        int* sizes = d_lengths + (idx*groups);
        int size = thrust::reduce(execpol->on(0), sizes, sizes+groups);
        if( size==0 )
            continue;
        char* d_buffer = device_alloc<char>(size,0);
        row->pImpl->setMemoryBuffer(d_buffer,size);
        strings[idx] = row->pImpl->getStringsPtr();
        buffers[idx] = d_buffer;
    }
    // copy each subgroup into each rows memory
    custring_view_array* d_rows = strings.data().get();
    char** d_buffers = buffers.data().get();
    if( (regex_insts > MAX_STACK_INSTS) || (regex_insts <= 10) )
        thrust::for_each_n(execpol->on(0), thrust::make_counting_iterator<unsigned int>(0), count,
            extract_record_fn<RX_STACK_SMALL>{prog, d_strings, groups, d_lengths, false, d_buffers, d_rows});
    else if( regex_insts <= 100 )
        thrust::for_each_n(execpol->on(0), thrust::make_counting_iterator<unsigned int>(0), count,
            extract_record_fn<RX_STACK_MEDIUM>{prog, d_strings, groups, d_lengths, false, d_buffers, d_rows});
    else
        thrust::for_each_n(execpol->on(0), thrust::make_counting_iterator<unsigned int>(0), count,
            extract_record_fn<RX_STACK_LARGE>{prog, d_strings, groups, d_lengths, false, d_buffers, d_rows});
    //
    hipError_t err = hipDeviceSynchronize();
    if( err != hipSuccess )
    {
        fprintf(stderr,"nvs-extract_record(%s): groups=%d\n",pattern,groups);
        printCudaError(err);
    }
    dreprog::destroy(prog);
    return groups;
}
