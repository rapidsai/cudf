/*
* Copyright (c) 2018-2019, NVIDIA CORPORATION.  All rights reserved.
*
* Licensed under the Apache License, Version 2.0 (the "License");
* you may not use this file except in compliance with the License.
* You may obtain a copy of the License at
*
*     http://www.apache.org/licenses/LICENSE-2.0
*
* Unless required by applicable law or agreed to in writing, software
* distributed under the License is distributed on an "AS IS" BASIS,
* WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
* See the License for the specific language governing permissions and
* limitations under the License.
*/

#include <hip/hip_runtime.h>
#include <thrust/device_vector.h>
#include <thrust/for_each.h>
#include <rmm/rmm.h>
#include <rmm/thrust_rmm_allocator.h>

#include "nvstrings/NVStrings.h"

#include "./NVStringsImpl.h"
#include "../custring_view.cuh"
#include "../util.h"

// remove the target characters from the beginning of each string
NVStrings* NVStrings::lstrip( const char* to_strip )
{
    unsigned int count = size();
    custring_view_array d_strings = pImpl->getStringsPtr();
    auto execpol = rmm::exec_policy(0);

    char* d_strip = nullptr;
    if( to_strip )
    {
        int len = (int)strlen(to_strip) + 1; // include null
        d_strip = device_alloc<char>(len,0);
        CUDA_TRY( hipMemcpyAsync(d_strip,to_strip,len,hipMemcpyHostToDevice))
    }

    // compute size of output buffer
    rmm::device_vector<size_t> lengths(count,0);
    size_t* d_lengths = lengths.data().get();
    thrust::for_each_n(execpol->on(0), thrust::make_counting_iterator<unsigned int>(0), count,
        [d_strings, d_strip, d_lengths] __device__(unsigned int idx){
            custring_view* dstr = d_strings[idx];
            if( !dstr )
                return;
            unsigned int len = dstr->lstrip_size(d_strip);
            len = ALIGN_SIZE(len);
            d_lengths[idx] = (size_t)len;
        });
    // create output object
    NVStrings* rtn = new NVStrings(count);
    char* d_buffer = rtn->pImpl->createMemoryFor(d_lengths);
    if( d_buffer==0 )
    {
        if( d_strip )
            RMM_FREE(d_strip,0);
        return rtn; // all strings are null
    }

    // create offsets
    rmm::device_vector<size_t> offsets(count,0);
    thrust::exclusive_scan(execpol->on(0),lengths.begin(),lengths.end(),offsets.begin());
    // do the strip
    custring_view** d_results = rtn->pImpl->getStringsPtr();
    size_t* d_offsets = offsets.data().get();
    thrust::for_each_n(execpol->on(0), thrust::make_counting_iterator<unsigned int>(0), count,
        [d_strings, d_strip, d_buffer, d_offsets, d_results] __device__(unsigned int idx){
            custring_view* dstr = d_strings[idx];
            if( !dstr )
                return;
            char* buffer = d_buffer + d_offsets[idx];
            d_results[idx] = dstr->lstrip(d_strip,buffer);
        });
    //
    if( d_strip )
        RMM_FREE(d_strip,0);
    return rtn;
}

// remove the target character from the beginning and the end of each string
NVStrings* NVStrings::strip( const char* to_strip )
{
    unsigned int count = size();
    custring_view_array d_strings = pImpl->getStringsPtr();
    auto execpol = rmm::exec_policy(0);

    char* d_strip = nullptr;
    if( to_strip )
    {
        int len = (int)strlen(to_strip) + 1; // include null
        d_strip = device_alloc<char>(len,0);
        CUDA_TRY( hipMemcpyAsync(d_strip,to_strip,len,hipMemcpyHostToDevice))
    }

    // compute size of output buffer
    rmm::device_vector<size_t> lengths(count,0);
    size_t* d_lengths = lengths.data().get();
    thrust::for_each_n(execpol->on(0), thrust::make_counting_iterator<unsigned int>(0), count,
        [d_strings, d_strip, d_lengths] __device__(unsigned int idx){
            custring_view* dstr = d_strings[idx];
            if( !dstr )
                return;
            unsigned int len = dstr->strip_size(d_strip);
            len = ALIGN_SIZE(len);
            d_lengths[idx] = (size_t)len;
        });
    // create output object
    NVStrings* rtn = new NVStrings(count);
    char* d_buffer = rtn->pImpl->createMemoryFor(d_lengths);
    if( d_buffer==0 )
    {
        if( d_strip )
            RMM_FREE(d_strip,0);
        return rtn;
    }

    // create offsets
    rmm::device_vector<size_t> offsets(count,0);
    thrust::exclusive_scan(execpol->on(0),lengths.begin(),lengths.end(),offsets.begin());
    // do the strip
    custring_view_array d_results = rtn->pImpl->getStringsPtr();
    size_t* d_offsets = offsets.data().get();
    thrust::for_each_n(execpol->on(0), thrust::make_counting_iterator<unsigned int>(0), count,
        [d_strings, d_strip, d_buffer, d_offsets, d_results] __device__(unsigned int idx){
            custring_view* dstr = d_strings[idx];
            if( !dstr )
                return;
            char* buffer = d_buffer + d_offsets[idx];
            d_results[idx] = dstr->strip(d_strip,buffer);
        });
    //
    if( d_strip )
        RMM_FREE(d_strip,0);
    return rtn;
}

// remove the target character from the end of each string
NVStrings* NVStrings::rstrip( const char* to_strip )
{
    unsigned int count = size();
    custring_view_array d_strings = pImpl->getStringsPtr();
    auto execpol = rmm::exec_policy(0);

    char* d_strip = nullptr;
    if( to_strip )
    {
        int len = (int)strlen(to_strip) + 1; // include null
        d_strip = device_alloc<char>(len,0);
        CUDA_TRY( hipMemcpyAsync(d_strip,to_strip,len,hipMemcpyHostToDevice))
    }

    // compute size of output buffer
    rmm::device_vector<size_t> lengths(count,0);
    size_t* d_lengths = lengths.data().get();
    thrust::for_each_n(execpol->on(0), thrust::make_counting_iterator<unsigned int>(0), count,
        [d_strings, d_strip, d_lengths] __device__(unsigned int idx){
            custring_view* dstr = d_strings[idx];
            if( !dstr )
                return;
            unsigned int len = dstr->rstrip_size(d_strip);
            len = ALIGN_SIZE(len);
            d_lengths[idx] = (size_t)len;
        });

    // create output object
    NVStrings* rtn = new NVStrings(count);
    char* d_buffer = rtn->pImpl->createMemoryFor(d_lengths);
    if( d_buffer==0 )
    {
        if( d_strip )
            RMM_FREE(d_strip,0);
        return rtn; // all strings are null
    }

    // create offsets
    rmm::device_vector<size_t> offsets(count,0);
    thrust::exclusive_scan(execpol->on(0),lengths.begin(),lengths.end(),offsets.begin());
    // do the strip
    custring_view_array d_results = rtn->pImpl->getStringsPtr();
    size_t* d_offsets = offsets.data().get();
    thrust::for_each_n(execpol->on(0), thrust::make_counting_iterator<unsigned int>(0), count,
        [d_strings, d_strip, d_buffer, d_offsets, d_results] __device__(unsigned int idx){
            custring_view* dstr = d_strings[idx];
            if( !dstr )
                return;
            char* buffer = d_buffer + d_offsets[idx];
            d_results[idx] = dstr->rstrip(d_strip,buffer);
        });
    //
    if( d_strip )
        RMM_FREE(d_strip,0);
    return rtn;
}
