#include "hip/hip_runtime.h"
/*
* Copyright (c) 2018-2019, NVIDIA CORPORATION.  All rights reserved.
*
* Licensed under the Apache License, Version 2.0 (the "License");
* you may not use this file except in compliance with the License.
* You may obtain a copy of the License at
*
*     http://www.apache.org/licenses/LICENSE-2.0
*
* Unless required by applicable law or agreed to in writing, software
* distributed under the License is distributed on an "AS IS" BASIS,
* WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
* See the License for the specific language governing permissions and
* limitations under the License.
*/

#include <exception>
#include <map>
#include <hip/hip_runtime.h>
#include <thrust/device_vector.h>
#include <thrust/for_each.h>
#include <thrust/count.h>
#include <rmm/rmm.h>
#include <rmm/thrust_rmm_allocator.h>

#include "nvstrings/NVStrings.h"

#include "./NVStringsImpl.h"
#include "../custring_view.cuh"
#include "../util.h"

//
// This is the functor for the url_encode() method below.
// Specific requirements are documented in custrings issue #321.
// In summary it converts mostly non-ascii characters and control characters into UTF-8 hex characters
// prefixed with '%'. For example, the space character must be converted to characters '%20' where the
// '20' indicates the hex value for space in UTF-8. Likewise, multi-byte characters are converted to
// multiple hex charactes. For example, the é character is converted to characters '%C3%A9' where 'C3A9'
// is the UTF-8 bytes xc3a9 for this character.
// Like other functors for NVStrings it is called twice.
// First to calculate the output string allocation size (bcompute_size_only=true).
// The 2nd call actually performs the encoding operation on the memory provided.
// 
struct url_encoder
{
    custring_view_array d_strings;
    size_t* d_offsets;
    bool bcompute_size_only{true};
    char* d_buffer{nullptr};
    custring_view_array d_results;

    // utility to create 2-byte hex characters from single binary byte
    __device__ void byte_to_hex( unsigned char byte, char* hex )
    {
        hex[0] = '0';
        if( byte >= 16 )
        {
            unsigned char hibyte = byte/16;
            hex[0] = hibyte < 10 ? '0'+hibyte : 'A'+(hibyte-10);
            byte = byte - (hibyte * 16);
        }
        hex[1] = byte < 10 ? '0'+byte : 'A'+(byte-10);
    }

    // main part of the functor the performs the url-encoding
    __device__ void operator()( unsigned int idx )
    {
        custring_view* dstr = d_strings[idx];
        if( !dstr )
            return;
        //
        char* buffer = d_buffer + d_offsets[idx];
        char* optr = buffer;
        int nbytes = 0;
        char hex[2]; // two-byte hex max
        for( auto itr = dstr->begin(); itr!=dstr->end(); itr++ )
        {
            Char ch = *itr;
            if( ch < 128 )
            {
                if( (ch>='0' && ch<='9') || // these are the characters
                    (ch>='A' && ch<='Z') || // that are not to be url encoded
                    (ch>='a' && ch<='z') || // reference: docs.python.org/3/library/urllib.parse.html#urllib.parse.quote
                    (ch=='.') || (ch=='_') || (ch=='~') || (ch=='-') )
                {
                    nbytes++;
                    if( !bcompute_size_only )
                    {
                        char* sptr = dstr->data() + itr.byte_offset();
                        copy_and_incr(optr,sptr,1);
                    }
                }
                else // url-encode everything else
                {
                    nbytes += 3;
                    if( !bcompute_size_only )
                    {
                        copy_and_incr(optr,(char*)"%",1);      // add the '%' prefix
                        byte_to_hex( (unsigned char)ch, hex);  // convert to 2 hex chars
                        copy_and_incr(optr,hex,2);             // add them to the output
                    }
                }
            }
            else // these are to be utf-8 url-encoded
            {
                unsigned char char_bytes[4]; // holds utf-8 bytes
                unsigned int char_width = custring_view::Char_to_char(ch,(char*)char_bytes);
                nbytes += char_width * 3; // '%' plus 2 hex chars per byte (example: é is %C3%A9)
                // process each byte in this current character
                for( unsigned int chidx=0; !bcompute_size_only && (chidx < char_width); chidx++ )
                {
                    copy_and_incr(optr,(char*)"%",1);     // add '%' prefix
                    byte_to_hex( char_bytes[chidx], hex); // convert to 2 hex chars
                    copy_and_incr(optr,hex,2);            // add them to the output
                }
            }
        }
        if( bcompute_size_only )
        {
            int size = custring_view::alloc_size(nbytes,nbytes);
            d_offsets[idx] = ALIGN_SIZE(size);
        }
        else
            d_results[idx] = custring_view::create_from(buffer,buffer,nbytes);
    }
};

// This method url-encodes each string and returns them in a new instance.
// See the functor above for detailed code logic executed for each string.
NVStrings* NVStrings::url_encode()
{
    auto execpol = rmm::exec_policy(0);
    unsigned int count = size();
    // inputs
    custring_view_array d_strings = pImpl->getStringsPtr();
    rmm::device_vector<size_t> offsets(count,0);
    size_t* d_offsets = offsets.data().get();
    // outputs
    NVStrings* rtn = nullptr;
    char* d_buffer = nullptr;
    custring_view_array d_results = nullptr;

    // first loop will compute size output
    // 2nd loop will do the operation in the allocated memory
    enum scan_and_operate { scan, operate };
    auto op = scan;
    while(true)
    {
        thrust::for_each_n(execpol->on(0), thrust::make_counting_iterator<unsigned int>(0), count,
            url_encoder{d_strings,d_offsets,(op==scan),d_buffer,d_results});
        if( op==operate )
            break;
        op = operate;
        rtn = new NVStrings(count);
        d_buffer = rtn->pImpl->createMemoryFor(d_offsets);
        if( !d_buffer )
            break; // all nulls, ok
        // create offsets
        thrust::exclusive_scan(execpol->on(0),offsets.begin(),offsets.end(),offsets.begin());
        d_results = rtn->pImpl->getStringsPtr();
    }
    //
    return rtn;
}

//
// This is the functor for the url_decode() method below.
// Specific requirements are documented in custrings issue #321.
// In summary it converts all character sequences starting with '%' into bytes
// interpretting the following 2 characters as hex values to create the output byte.
// For example, the sequence '%20' is converted into byte (0x20) which is a single
// space character. Another example converts '%C3%A9' into 2 sequential bytes
// (0xc3 and 0xa9 respectively). Overall, 3 characters are converted into one byte
// whenever a '%' character is encountered in the string.
// Like other functors for NVStrings it is called twice.
// First to calculate the output string allocation size (bcompute_size_only=true).
// The 2nd call actually performs the operation on the memory provided.
// 
struct url_decoder
{
    custring_view_array d_strings;
    size_t* d_offsets;
    bool bcompute_size_only{true};
    char* d_buffer;
    custring_view_array d_results;

    // utility to convert 2 hex chars into a single byte
    __device__ char hex_to_byte( char ch1, char ch2 )
    {
        unsigned char result = 0;
        if( ch1 >= '0' && ch1 <= '9' )
            result += (ch1-48);
        else if( ch1 >= 'A' && ch1 <= 'Z' )
            result += (ch1-55);
        else if( ch1 >='a' && ch1 <= 'z' )
            result += (ch1-87);
        result *= 16;
        if( ch2 >= '0' && ch2 <= '9' )
            result += (ch2-48);
        else if( ch2 >= 'A' && ch2 <= 'Z' )
            result += (ch2-55);
        else if( ch2 >='a' && ch2 <= 'z' )
            result += (ch2-87);
        return (char)result;
    }

    // main functor method executed on each string
    __device__ void operator()(unsigned int idx)
    {
        custring_view* dstr = d_strings[idx];
        if( !dstr )
            return;
        char* buffer = d_buffer + d_offsets[idx];
        char* optr = buffer;
        unsigned int nbytes = 0, nchars = 0;
        char* sptr = dstr->data();
        char* send = sptr + dstr->size();
        while( sptr < send ) // walk through each byte
        {
            char ch = *sptr++;
            if( (ch == '%') && ((sptr+1) < send) )
            {   // found '%', convert hex to byte
                ch = *sptr++;
                ch = hex_to_byte( ch, *sptr++ );
            }
            ++nbytes; // keeping track of bytes and chars
            nchars += int((((unsigned char)(ch)) & 0xC0) != 0x80); // utf8 ext byte
            if( !bcompute_size_only )
                copy_and_incr(optr, &ch, 1);
        }
        if( bcompute_size_only )
        {
            int size = custring_view::alloc_size(nbytes,nchars);
            d_offsets[idx] = ALIGN_SIZE(size);
        }
        else
            d_results[idx] = custring_view::create_from(buffer,buffer,nbytes);
    }
};

// This method url-decodes each string and returns them in a new instance.
// See the functor above for the detailed code logic executed for each string.
NVStrings* NVStrings::url_decode()
{
    auto execpol = rmm::exec_policy(0);
    unsigned int count = size();
    // inputs
    custring_view_array d_strings = pImpl->getStringsPtr();
    rmm::device_vector<size_t> offsets(count,0);
    size_t* d_offsets = offsets.data().get();
    // outputs
    NVStrings* rtn = nullptr;
    char* d_buffer = nullptr;
    custring_view_array d_results = nullptr;

    // first loop will compute size output
    // 2nd loop will do the operation in the allocated memory
    enum scan_and_operate { scan, operate };
    auto op = scan;
    while(true)
    {
        thrust::for_each_n(execpol->on(0), thrust::make_counting_iterator<unsigned int>(0), count,
            url_decoder{d_strings,d_offsets,(op==scan),d_buffer,d_results});
        if( op==operate )
            break;
        op = operate;
        rtn = new NVStrings(count);
        d_buffer = rtn->pImpl->createMemoryFor(d_offsets);
        if( !d_buffer )
            break; // all nulls, ok
        // create offsets
        thrust::exclusive_scan(execpol->on(0),offsets.begin(),offsets.end(),offsets.begin());
        d_results = rtn->pImpl->getStringsPtr();
    }
    //
    return rtn;
}

