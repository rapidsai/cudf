/*
* Copyright (c) 2018-2019, NVIDIA CORPORATION.  All rights reserved.
*
* Licensed under the Apache License, Version 2.0 (the "License");
* you may not use this file except in compliance with the License.
* You may obtain a copy of the License at
*
*     http://www.apache.org/licenses/LICENSE-2.0
*
* Unless required by applicable law or agreed to in writing, software
* distributed under the License is distributed on an "AS IS" BASIS,
* WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
* See the License for the specific language governing permissions and
* limitations under the License.
*/

#include <exception>
#include <hip/hip_runtime.h>
#include <thrust/device_vector.h>
#include <thrust/for_each.h>
#include <rmm/rmm.h>
#include <rmm/thrust_rmm_allocator.h>
#include <cudf/utilities/error.hpp>

#include "nvstrings/NVStrings.h"

#include "./NVStringsImpl.h"
#include "../custring_view.cuh"
#include "../util.h"

//
NVStrings* NVStrings::cat( NVStrings* others, const char* separator, const char* narep )
{
    if( others==0 )
        return nullptr; // return a copy of ourselves?
    unsigned int count = size();
    if( others->size() != count )
        throw std::invalid_argument("nvstrings::cat sizes do not match");

    auto execpol = rmm::exec_policy(0);
    unsigned int seplen = 0;
    if( separator )
        seplen = (unsigned int)strlen(separator);
    char* d_sep = nullptr;
    if( seplen )
    {
        d_sep = device_alloc<char>(seplen,0);
        CUDA_TRY( hipMemcpyAsync(d_sep,separator,seplen,hipMemcpyHostToDevice))
    }
    unsigned int narlen = 0;
    char* d_narep = nullptr;
    if( narep )
    {
        narlen = (unsigned int)strlen(narep);
        d_narep = device_alloc<char>(narlen+1,0);
        CUDA_TRY( hipMemcpyAsync(d_narep,narep,narlen+1,hipMemcpyHostToDevice))
    }

    custring_view_array d_strings = pImpl->getStringsPtr();
    custring_view_array d_others = others->pImpl->getStringsPtr();

    // first compute the size of the output
    rmm::device_vector<size_t> sizes(count,0);
    size_t* d_sizes = sizes.data().get();
    thrust::for_each_n(execpol->on(0), thrust::make_counting_iterator<unsigned int>(0), count,
        [d_strings, d_others, d_sep, seplen, d_narep, narlen, d_sizes] __device__(unsigned int idx){
            custring_view* dstr1 = d_strings[idx];
            custring_view* dstr2 = d_others[idx];
            if( (!dstr1 || !dstr2) && !d_narep )
                return; // null case
            int nchars = 0;
            int bytes = 0;
            // left side
            if( dstr1 )
            {
                nchars = dstr1->chars_count();
                bytes = dstr1->size();
            }
            else if( d_narep )
            {
                nchars = custring_view::chars_in_string(d_narep,narlen);
                bytes = narlen;
            }
            // separator
            if( d_sep )
            {
                nchars += custring_view::chars_in_string(d_sep,seplen);
                bytes += seplen;
            }
            // right side
            if( dstr2 )
            {
                nchars += dstr2->chars_count();
                bytes += dstr2->size();
            }
            else if( d_narep )
            {
                nchars += custring_view::chars_in_string(d_narep,narlen);
                bytes += narlen;
            }
            int size = custring_view::alloc_size(bytes,nchars);
            //printf("cat:%lu:size=%d\n",idx,size);
            size = ALIGN_SIZE(size);
            d_sizes[idx] = size;
        });

    // allocate the memory for the output
    NVStrings* rtn = new NVStrings(count);
    char* d_buffer = rtn->pImpl->createMemoryFor(d_sizes);
    if( d_buffer==0 )
    {
        if( d_sep )
            RMM_FREE(d_sep,0);
        if( d_narep )
            RMM_FREE(d_narep,0);
        return rtn;
    }
    CUDA_TRY(hipMemset(d_buffer,0,rtn->pImpl->getMemorySize()));
    // compute the offset
    rmm::device_vector<size_t> offsets(count,0);
    thrust::exclusive_scan(execpol->on(0),sizes.begin(),sizes.end(),offsets.begin());
    // do the thing
    custring_view_array d_results = rtn->pImpl->getStringsPtr();
    size_t* d_offsets = offsets.data().get();
    thrust::for_each_n(execpol->on(0), thrust::make_counting_iterator<unsigned int>(0), count,
        [d_strings, d_others, d_sep, seplen, d_narep, narlen, d_buffer, d_offsets, d_results] __device__(unsigned int idx){
            char* buffer = d_buffer + d_offsets[idx];
            custring_view* dstr1 = d_strings[idx];
            custring_view* dstr2 = d_others[idx];
            if( (!dstr1 || !dstr2) && !d_narep )
                return; // if null, an no null rep, done
            custring_view* dout = custring_view::create_from(buffer,0,0); // init empty string
            if( dstr1 )
                dout->append(*dstr1);        // add left side
            else if( d_narep )               // (or null rep)
                dout->append(d_narep,narlen);
            if( d_sep )
                dout->append(d_sep,seplen);  // add separator
            if( dstr2 )
                dout->append(*dstr2);        // add right side
            else if( d_narep )               // (or null rep)
                dout->append(d_narep,narlen);
            //printf("cat:%lu:[]=%d\n",idx,dout->size());
            d_results[idx] = dout;
    });
    //printCudaError(hipDeviceSynchronize(),"nvs-cat: combining strings");

    if( d_sep )
        RMM_FREE(d_sep,0);
    if( d_narep )
        RMM_FREE(d_narep,0);
    return rtn;
}

//
NVStrings* NVStrings::cat( std::vector<NVStrings*>& others, const char* separator, const char* narep )
{
    if( others.size()==0 )
        return nullptr; // return a copy of ourselves?
    unsigned int count = size();
    for( auto itr=others.begin(); itr!=others.end(); itr++ )
        if( (*itr)->size() != count )
            throw std::invalid_argument("nvstrings::cat sizes do not match");

    auto execpol = rmm::exec_policy(0);
    custring_view* d_separator = nullptr;
    if( separator )
    {
        unsigned int seplen = (unsigned int)strlen(separator);
        unsigned int sep_size = custring_view::alloc_size(separator,seplen);
        d_separator = reinterpret_cast<custring_view*>(device_alloc<char>(sep_size,0));
        custring_view::create_from_host(d_separator,separator,seplen);
    }
    custring_view* d_narep = nullptr;
    if( narep )
    {
        unsigned int narlen = (unsigned int)strlen(narep);
        unsigned int nar_size = custring_view::alloc_size(narep,narlen);
        d_narep = reinterpret_cast<custring_view*>(device_alloc<char>(nar_size,0));
        custring_view::create_from_host(d_narep,narep,narlen);
    }

    custring_view_array d_strings = pImpl->getStringsPtr();
    rmm::device_vector<custring_view_array> dothers;
    for( auto itr=others.begin(); itr!=others.end(); itr++ )
        dothers.push_back((*itr)->pImpl->getStringsPtr());
    custring_view_array* d_others = dothers.data().get();
    unsigned int others_count = (unsigned int)others.size();

    // first compute the size of the output
    rmm::device_vector<size_t> sizes(count,0);
    size_t* d_sizes = sizes.data().get();
    thrust::for_each_n(execpol->on(0), thrust::make_counting_iterator<unsigned int>(0), count,
        [d_strings, d_others, others_count, d_separator, d_narep, d_sizes] __device__(unsigned int idx){
            custring_view* dstr = d_strings[idx];
            int nchars = 0;
            int bytes = 0;
            bool allnulls = !dstr && !d_narep;
            if( dstr )
            {
                nchars += dstr->chars_count();
                bytes += dstr->size();
            }
            else if( d_narep )
            {
                nchars += d_narep->chars_count();
                bytes += d_narep->size();
            }
            for( unsigned int jdx=0; !allnulls && (jdx < others_count); ++jdx )
            {
                custring_view_array dcat2 = d_others[jdx];
                dstr = dcat2[idx];
                allnulls = !dstr && !d_narep;
                if( d_separator )
                {
                    nchars += d_separator->chars_count();
                    bytes += d_separator->size();
                }
                if( dstr )
                {
                    nchars += dstr->chars_count();
                    bytes += dstr->size();
                }
                else if( d_narep )
                {
                    nchars += d_narep->chars_count();
                    bytes += d_narep->size();
                }
            }
            int size = custring_view::alloc_size(bytes,nchars);
            size = ALIGN_SIZE(size);
            if( allnulls )
                size = 0;
            //printf("cat:%lu:size=%d\n",idx,size);
            d_sizes[idx] = size;
        });

    // allocate the memory for the output
    NVStrings* rtn = new NVStrings(count);
    char* d_buffer = rtn->pImpl->createMemoryFor(d_sizes);
    if( d_buffer==0 )
    {
        if( d_separator )
            RMM_FREE(d_separator,0);
        if( d_narep )
            RMM_FREE(d_narep,0);
        return rtn;
    }
    CUDA_TRY(hipMemset(d_buffer,0,rtn->pImpl->getMemorySize()));
    // compute the offset
    rmm::device_vector<size_t> offsets(count,0);
    thrust::exclusive_scan(execpol->on(0),sizes.begin(),sizes.end(),offsets.begin());
    // do the thing
    custring_view_array d_results = rtn->pImpl->getStringsPtr();
    size_t* d_offsets = offsets.data().get();
    thrust::for_each_n(execpol->on(0), thrust::make_counting_iterator<unsigned int>(0), count,
        [d_strings, d_others, others_count, d_separator, d_narep, d_buffer, d_sizes, d_offsets, d_results] __device__(unsigned int idx){
            if( d_sizes[idx]==0 )
                return; // null string
            char* buffer = d_buffer + d_offsets[idx];
            custring_view* dstr = d_strings[idx];
            custring_view* dout = custring_view::create_from(buffer,0,0); // init empty string
            if( dstr )
                dout->append(*dstr);
            else if( d_narep )
                dout->append(*d_narep);
            for( unsigned int jdx=0; jdx < others_count; ++jdx )
            {
                custring_view_array dcat2 = d_others[jdx];
                dstr = dcat2[idx];
                if( d_separator )
                    dout->append(*d_separator);
                if( dstr )
                    dout->append(*dstr);
                else if( d_narep )
                    dout->append(*d_narep);
            }
            //printf("cat:%lu:[]=%d\n",idx,dout->size());
            d_results[idx] = dout;
    });
    //printCudaError(hipDeviceSynchronize(),"nvs-cat: combining strings");

    if( d_separator )
        RMM_FREE(d_separator,0);
    if( d_narep )
        RMM_FREE(d_narep,0);
    return rtn;
}

// this returns one giant string joining all the strings
// in the list with the delimiter string between each one
NVStrings* NVStrings::join( const char* delimiter, const char* narep )
{
    if( delimiter==0 )
        throw std::invalid_argument("nvstrings::join delimiter cannot be null");
    auto execpol = rmm::exec_policy(0);
    unsigned int dellen = (unsigned int)strlen(delimiter);
    char* d_delim = nullptr;
    if( dellen > 0 )
    {
        d_delim = device_alloc<char>(dellen,0);
        CUDA_TRY( hipMemcpyAsync(d_delim,delimiter,dellen,hipMemcpyHostToDevice))
    }
    unsigned int narlen = 0;
    char* d_narep = nullptr;
    if( narep )
    {
        narlen = (unsigned int)strlen(narep);
        d_narep = device_alloc<char>(narlen+1,0);
        CUDA_TRY( hipMemcpyAsync(d_narep,narep,narlen+1,hipMemcpyHostToDevice))
    }

    unsigned int count = size();
    custring_view** d_strings = pImpl->getStringsPtr();

    // need to compute the giant buffer size
    rmm::device_vector<size_t> lens(count,0);
    size_t* d_lens = lens.data().get();
    rmm::device_vector<size_t> chars(count,0);
    size_t* d_chars = chars.data().get();
    thrust::for_each_n(execpol->on(0), thrust::make_counting_iterator<unsigned int>(0), count,
        [d_strings, d_delim, dellen, d_narep, narlen, count, d_lens, d_chars] __device__(unsigned int idx){
            custring_view* dstr = d_strings[idx];
            int dlen = dellen;
            int nchars = 0;
            int bytes = 0;
            if( idx+1 >= count )
                dlen = 0; // no trailing delimiter
            if( dstr )
            {
                nchars = dstr->chars_count();
                bytes = dstr->size();
            }
            else if( d_narep )
            {
                nchars = custring_view::chars_in_string(d_narep,narlen);
                bytes = narlen;
            }
            else
                dlen = 0; // for null, no delimiter
            if( dlen )
            {
                nchars += custring_view::chars_in_string(d_delim,dellen);
                bytes += dellen;
            }
            d_lens[idx] = bytes;
            d_chars[idx] = nchars;
        });

    //hipDeviceSynchronize();
    // compute how much space is required for the giant string
    size_t totalBytes = thrust::reduce(execpol->on(0), lens.begin(), lens.end());
    size_t totalChars = thrust::reduce(execpol->on(0), chars.begin(), chars.end());
    //printf("totalBytes=%ld, totalChars=%ld\n",totalBytes,totalChars);
    size_t allocSize = custring_view::alloc_size((unsigned int)totalBytes,(unsigned int)totalChars);
    //printf("allocSize=%ld\n",allocSize);

    // convert the lens values into offsets
    rmm::device_vector<size_t> offsets(count,0);
    thrust::exclusive_scan(lens.begin(),lens.end(),offsets.begin());
    size_t* d_offsets = offsets.data().get();
    // create one big buffer to hold the strings
    char* d_buffer = nullptr;
    rmmError_t rmmerr = RMM_ALLOC(&d_buffer,allocSize,0);
    if( rmmerr != RMM_SUCCESS )
    {
        std::ostringstream message;
        message << "allocate error " << rmmerr;
        throw std::runtime_error(message.str());
    }
    NVStrings* rtn = new NVStrings(1);
    custring_view_array d_result = rtn->pImpl->getStringsPtr();
    rtn->pImpl->setMemoryBuffer(d_buffer,allocSize);
    // copy the strings into it
    thrust::for_each_n(execpol->on(0), thrust::make_counting_iterator<unsigned int>(0), count,
        [d_strings, d_buffer, d_offsets, count, d_delim, dellen, d_narep, narlen] __device__(unsigned int idx){
            char* sptr = d_buffer + 8 + d_offsets[idx];
            char* dlim = d_delim;
            custring_view* dstr = d_strings[idx];
            if( dstr )
            {   // copy string to output
                int ssz = dstr->size();
                memcpy(sptr,dstr->data(),ssz);
                sptr += ssz;
            }
            else if( d_narep )
            {   // or copy null-replacement to output
                memcpy(sptr,d_narep,narlen);
                sptr += narlen;
            }
            else // or copy nothing to output
                dlim = 0; // prevent delimiter copy below
            // copy delimiter to output
            if( (idx+1 < count) && dlim )
                memcpy(sptr,dlim,dellen);
        });

    // assign to resulting custring_view
    thrust::for_each_n(execpol->on(0), thrust::make_counting_iterator<unsigned int>(0), 1,
        [d_buffer, totalBytes, d_result] __device__ (unsigned int idx){
            char* sptr = d_buffer + 8;
            d_result[0] = custring_view::create_from(d_buffer,sptr,totalBytes);
        });
    //printCudaError(hipDeviceSynchronize(),"nvs-join");

    if( d_delim )
        RMM_FREE(d_delim,0);
    if( d_narep )
        RMM_FREE(d_narep,0);
    return rtn;
}
