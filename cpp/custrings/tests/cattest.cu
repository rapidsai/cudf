#include <memory>
#include <string.h>
#include <stdlib.h>
#include <string>
#include <vector>
#include <hip/hip_runtime.h>
#include <thrust/execution_policy.h>
#include <thrust/device_vector.h>
#include <thrust/for_each.h>

#include "nvstrings/NVStrings.h"
#include "nvstrings/NVCategory.h"

//
// cd ../cpp/build
// nvcc -w -std=c++11 --expt-extended-lambda -gencode arch=compute_70,code=sm_70 ../tests/cattest.cu -L. -lNVStrings -lNVCategory -o cattest --linker-options -rpath,.:
//

// csv file contents in device memory
void* d_fileContents = 0;

// return a vector of DString's we wish to process
std::pair<const char*,size_t>* setupTest(int& linesCount, int column)
{
    FILE* fp = fopen("../../data/36634-rows.csv", "rb");
    if( !fp )
    {
        printf("missing csv file\n");
        return 0;
    }
	fseek(fp, 0, SEEK_END);
	int fileSize = (int)ftell(fp);
    fseek(fp, 0, SEEK_SET);
    printf("File size = %d bytes\n", fileSize);
    if( fileSize < 2 )
    {
        fclose(fp);
        return 0;
    }
    // load file into memory
    int contentsSize = fileSize+2;
    char* contents = new char[contentsSize+2];
    fread(contents, 1, fileSize, fp);
    contents[fileSize] = '\r'; // line terminate
	contents[fileSize+1] = 0;  // and null-terminate
	fclose(fp);

    // find lines -- compute offsets vector values
    thrust::host_vector<int> lineOffsets;
    char* ptr = contents;
    while( *ptr )
    {
        char ch = *ptr;
        if( ch=='\r' )
        {
            *ptr = 0;
            while(ch && (ch < ' ')) ch = *(++ptr);
            lineOffsets.push_back((int)(ptr - contents));
            continue;
        }    
        ++ptr;
    }
    linesCount = (int)lineOffsets.size();
    printf("Found %d lines\n",linesCount);
    // copy file contents into device memory
    char* d_contents = 0;
    hipMalloc(&d_contents,contentsSize);
    hipMemcpy(d_contents,contents,contentsSize,hipMemcpyHostToDevice);
    delete contents; // done with the host data

    // copy offsets vector into device memory
    thrust::device_vector<int> offsets(lineOffsets);
    int* d_offsets = offsets.data().get();
    // build empty output vector of DString*'s
    --linesCount; // removed header line
    std::pair<const char*,size_t>* d_column1 = 0;
    hipMalloc(&d_column1, linesCount * sizeof(std::pair<const char*,size_t>));

    // create a vector of DStrings using the first column of each line
    thrust::for_each_n(thrust::device,
      thrust::make_counting_iterator<size_t>(0), linesCount, 
      [d_contents, d_offsets, column, d_column1] __device__(size_t idx){
        // probably some more elegant way to do this
        int lineOffset = d_offsets[idx];
        int lineLength = d_offsets[idx+1] - lineOffset;
        d_column1[idx].first = (const char*)0;
        if( lineLength < 1 )
            return;
        char* line = &(d_contents[lineOffset]);
        char* stringStart = line;
        int columnLength = 0, col = 0;
        for( int i=0; (i < lineLength); ++i )
        {
            if( line[i] && line[i] != ',' )
            {
                ++columnLength;
                continue;
            }
            if( col++ >= column )
                break;
            stringStart = line + i + 1;
            columnLength = 0;
        }
        if( columnLength < 1 ) 
            return;
        // add string to vector array
        d_column1[idx].first = (const char*)stringStart;
        d_column1[idx].second = (size_t)columnLength;
      });
    //
    hipDeviceSynchronize();
    d_fileContents = d_contents;
    return d_column1;
}

int main( int argc, char** argv )
{
    //NVStrings::initLibrary();

    int count = 0;
    std::pair<const char*,size_t>* column1 = setupTest(count,16);
    if( column1==0 )
        return -1;

    NVStrings* dstrs = NVStrings::create_from_index( column1, count );

    hipFree(d_fileContents); // csv data not needed once dstrs is created
    hipFree(column1);        // string index data has done its job as well

    //
    int basize = (count+7)/8;
    unsigned char* d_bitarray = new unsigned char[basize];
    int ncount = dstrs->set_null_bitarray(d_bitarray,false,false);
    printf("str: null count = %d/%d\n",ncount,count);

    NVCategory* dcat = NVCategory::create_from_strings(*dstrs);
    printf("number of keys = %u\n", dcat->keys_size());
    printf("number of values = %u\n", dcat->size());
    unsigned char* d_bitarray2 = new unsigned char[basize];
    ncount = dcat->set_null_bitarray(d_bitarray2,false);
    printf("cat: null count = %d/%d\n",ncount,count);
    fflush(0);

    delete d_bitarray;
    delete d_bitarray2;

    NVStrings::destroy(dstrs);
    dstrs = dcat->get_keys();
    printf("keys:\n");
    dstrs->print();
    std::pair<int,int> bounds = dcat->get_value_bounds("Plastic");
    printf("Plastic would go (%d,%d)\n",bounds.first,bounds.second);
    bounds = dcat->get_value_bounds("Wood");
    printf("Wood is at (%d,%d)\n",bounds.first,bounds.second);
    bounds = dcat->get_value_bounds("Artisan");
    printf("Artisan (%d,%d)\n",bounds.first,bounds.second);
    bounds = dcat->get_value_bounds("Zebra");
    printf("Zebra (%d,%d)\n",bounds.first,bounds.second);

    NVCategory::destroy(dcat);
    NVStrings::destroy(dstrs);
    return 0;
}