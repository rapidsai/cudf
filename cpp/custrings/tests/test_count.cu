#include <gtest/gtest.h>
#include <vector>
#include <string>
#include <thrust/device_vector.h>

#include "nvstrings/NVStrings.h"

std::vector<const char*> hstrs{
        "The quick brown @fox jumps", "ovér the", "lazy @dog",
        "1234", "00:0:00", nullptr, "" };

TEST(TestCount, Contains)
{
    NVStrings* strs = NVStrings::create_from_array(hstrs.data(), hstrs.size());

    thrust::device_vector<bool> results(hstrs.size(),false);

    {
        strs->contains("é", results.data().get());
        bool expected[] = { false, true, false, false, false, false, false };
        for( int idx = 0; idx < (int) hstrs.size(); ++idx )
            EXPECT_EQ(results[idx],expected[idx]);
    }

    {
        strs->contains_re("\\d+", results.data().get());
        bool expected[] = { false, false, false, true, true, false, false };
        for( int idx = 0; idx < (int) hstrs.size(); ++idx )
            EXPECT_EQ(results[idx],expected[idx]);
    }

    {
        strs->contains_re("@\\w+", results.data().get());
        bool expected[] = { true, false, true, false, false, false, false };
        for( int idx = 0; idx < (int) hstrs.size(); ++idx )
            EXPECT_EQ(results[idx],expected[idx]);
    }

    NVStrings::destroy(strs);
}

TEST(TestCount, Match)
{
    NVStrings* strs = NVStrings::create_from_array(hstrs.data(), hstrs.size());

    thrust::device_vector<bool> results(hstrs.size(),false);

    {
        strs->match("ov[eé]r", results.data().get());
        bool expected[] = { false, true, false, false, false, false, false };
        for( int idx = 0; idx < (int) hstrs.size(); ++idx )
            EXPECT_EQ(results[idx],expected[idx]);
    }

    {
        strs->match("[tT]he", results.data().get());
        bool expected[] = { true, false, false, false, false, false, false };
        for( int idx = 0; idx < (int) hstrs.size(); ++idx )
            EXPECT_EQ(results[idx],expected[idx]);
    }

    {
        strs->match("\\d+", results.data().get());
        bool expected[] = { false, false, false, true, true, false, false };
        for( int idx = 0; idx < (int) hstrs.size(); ++idx )
            EXPECT_EQ(results[idx],expected[idx]);
    }

    NVStrings::destroy(strs);
}

TEST(TestCount, Count)
{
    NVStrings* strs = NVStrings::create_from_array(hstrs.data(), hstrs.size());

    thrust::device_vector<int> results(hstrs.size(),0);

    {
        strs->count_re("[tT]he", results.data().get());
        int expected[] = { 1, 1, 0,0,0,0 };
        for( int idx = 0; idx < (int) hstrs.size(); ++idx )
            EXPECT_EQ(results[idx],expected[idx]);
    }

    {
        strs->count_re("@\\w+", results.data().get());
        int expected[] = { 1, 0, 1, 0,0,0,0 };
        for( int idx = 0; idx < (int) hstrs.size(); ++idx )
            EXPECT_EQ(results[idx],expected[idx]);
    }

    {
        strs->count_re("\\d+:\\d+", results.data().get());
        int expected[] = { 0,0,0, 0,1,0,0 };
        for( int idx = 0; idx < (int) hstrs.size(); ++idx )
            EXPECT_EQ(results[idx],expected[idx]);
    }

    NVStrings::destroy(strs);
}


int main( int argc, char** argv )
{
    testing::InitGoogleTest(&argc,argv);
    return RUN_ALL_TESTS();
}