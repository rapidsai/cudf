#include <gtest/gtest.h>
#include <vector>
#include <hip/hip_runtime.h>
#include <thrust/device_vector.h>

#include "nvstrings/NVStrings.h"

#include "./utils.h"

TEST(TestTimestamp, ToTimestamp)
{
    {    
        std::vector<const char*> hstrs{"1974-02-28T01:23:45Z", "2019-07-17T21:34:37Z",
                                       nullptr, "" };
        NVStrings* strs = NVStrings::create_from_array(hstrs.data(),hstrs.size());
        thrust::device_vector<unsigned long> results(hstrs.size(),0);
        strs->timestamp2long("%Y-%m-%dT%H:%M:%SZ", NVStrings::seconds, results.data().get());
        int expected[] = { 131246625, 1563399277, 0,0 };
        for( int idx = 0; idx < (int) hstrs.size(); ++idx )
            EXPECT_EQ((int)results[idx],expected[idx]);
        NVStrings::destroy(strs);
    }

    {    
        std::vector<const char*> hstrs{"12.28.1982", "07.17.2019" };
        NVStrings* strs = NVStrings::create_from_array(hstrs.data(),hstrs.size());
        thrust::device_vector<unsigned long> results(hstrs.size(),0);
        strs->timestamp2long("%m-%d-%Y", NVStrings::days, results.data().get());
        int expected[] = { 4744, 18094 };
        for( int idx = 0; idx < (int) hstrs.size(); ++idx )
            EXPECT_EQ((int)results[idx],expected[idx]);
        NVStrings::destroy(strs);
    }
}

TEST(TestTimestamp, FromTimestamp)
{
    {    
        unsigned long values[] = {1563399273};
        thrust::device_vector<unsigned long> results(1);
        hipMemcpy( results.data().get(), values, 1*sizeof(unsigned long), hipMemcpyHostToDevice);
        NVStrings* got = NVStrings::long2timestamp(results.data().get(),1,NVStrings::seconds,"%m/%d/%Y %H:%M");
        const char* expected[] = { "07/17/2019 21:34" };
        EXPECT_TRUE( verify_strings(got,expected));
        NVStrings::destroy(got);
    }
    {    
        unsigned long values[] = {1563399273123};
        thrust::device_vector<unsigned long> results(1);
        hipMemcpy( results.data().get(), values, 1*sizeof(unsigned long), hipMemcpyHostToDevice);
        NVStrings* got = NVStrings::long2timestamp(results.data().get(),1,NVStrings::ms,"%H:%M:%S.%f");
        const char* expected[] = { "21:34:33.123" };
        EXPECT_TRUE( verify_strings(got,expected));
        NVStrings::destroy(got);
    }
}


int main( int argc, char** argv )
{
    testing::InitGoogleTest(&argc,argv);
    return RUN_ALL_TESTS();
}