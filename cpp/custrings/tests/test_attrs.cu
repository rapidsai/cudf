#include <gtest/gtest.h>
#include <vector>
#include <thrust/device_vector.h>

#include "nvstrings/NVStrings.h"

std::vector<const char*> hstrs{
        "Héllo", "thesé", nullptr, "ARE THE", "tést strings", "",
        "1.75", "-34", "+9.8", "17¼", "x³", "2³", " 12⅝",
        "1234567890", "de", "\t\r\n\f "};

TEST(TestAttrs, CharCounts)
{
    NVStrings* strs = NVStrings::create_from_array(hstrs.data(), hstrs.size());

    thrust::device_vector<int> lengths(hstrs.size(),0);
    strs->len(lengths.data().get());

    int expected[] = { 5, 5, -1, 7, 12, 0, 4, 3, 4, 3, 2, 2, 4, 10, 2, 5};
    for( int idx = 0; idx < (int) hstrs.size(); ++idx )
        EXPECT_EQ(lengths[idx],expected[idx]);

    NVStrings::destroy(strs);
}

TEST(TestAttrs, ByteCounts)
{
    NVStrings* strs = NVStrings::create_from_array(hstrs.data(), hstrs.size());

    thrust::device_vector<int> lengths(hstrs.size(),0);
    strs->byte_count(lengths.data().get());

    int expected[] = { 6, 6, -1, 7, 13, 0, 4, 3, 4, 4, 3, 3, 6, 10, 2, 5};
    for( int idx = 0; idx < (int) hstrs.size(); ++idx )
        EXPECT_EQ(lengths[idx],expected[idx]);

    NVStrings::destroy(strs);
}

TEST(TestAttrs, IsAlpha)
{
    NVStrings* strs = NVStrings::create_from_array(hstrs.data(), hstrs.size());

    thrust::device_vector<bool> results(hstrs.size(),false);
    strs->isalnum(results.data().get());
    {
        bool expected[] = { true, true, false, false, false, false,
                            false, false, false, true, true, true, false,
                            true, true, false };
        for( int idx = 0; idx < (int) hstrs.size(); ++idx )
            EXPECT_EQ(results[idx],expected[idx]);
    }

    strs->isalpha(results.data().get());
    {
        bool expected[] = { true, true, false, false, false, false,
                            false, false, false, false, false, false, false,
                            false, true, false };
        for( int idx = 0; idx < (int) hstrs.size(); ++idx )
            EXPECT_EQ(results[idx],expected[idx]);
    }

    strs->isspace(results.data().get());
    {
        bool expected[] = { false, false, false, false, false, false,
                            false, false, false, false, false, false, false,
                            false, false, true };
        for( int idx = 0; idx < (int) hstrs.size(); ++idx )
            EXPECT_EQ(results[idx],expected[idx]);
    }

    NVStrings::destroy(strs);
}

TEST(TestAttrs, IsNumeric)
{
    NVStrings* strs = NVStrings::create_from_array(hstrs.data(), hstrs.size());

    thrust::device_vector<bool> results(hstrs.size(),false);
    strs->isdigit(results.data().get());
    {
        bool expected[] = { false, false, false, false, false, false,
                            false, false, false, false, false, true, false,
                            true, false, false };
        for( int idx = 0; idx < (int) hstrs.size(); ++idx )
            EXPECT_EQ(results[idx],expected[idx]);
    }

    strs->isdecimal(results.data().get());
    {
        bool expected[] = { false, false, false, false, false, false,
                            false, false, false, false, false, false, false,
                            true, false, false };
        for( int idx = 0; idx < (int) hstrs.size(); ++idx )
            EXPECT_EQ(results[idx],expected[idx]);
    }

    strs->isnumeric(results.data().get());
    {
        bool expected[] = { false, false, false, false, false, false,
                            false, false, false, true, false, true, false,
                            true, false, false };
        for( int idx = 0; idx < (int) hstrs.size(); ++idx )
            EXPECT_EQ(results[idx],expected[idx]);
    }

    NVStrings::destroy(strs);
}

TEST(TestAttrs, IsSpace)
{
    NVStrings* strs = NVStrings::create_from_array(hstrs.data(), hstrs.size());

    thrust::device_vector<bool> results(hstrs.size(),false);

    strs->isspace(results.data().get());
    {
        bool expected[] = { false, false, false, false, false, false,
                            false, false, false, false, false, false, false,
                            false, false, true };
        for( int idx = 0; idx < (int) hstrs.size(); ++idx )
            EXPECT_EQ(results[idx],expected[idx]);
    }

    strs->is_empty(results.data().get());
    {
        bool expected[] = { false, false, true, false, false, true,
                            false, false, false, false, false, false, false,
                            false, false, false };
        for( int idx = 0; idx < (int) hstrs.size(); ++idx )
            EXPECT_EQ(results[idx],expected[idx]);
    }

    NVStrings::destroy(strs);
}

TEST(TestAttrs, IsUpperLower)
{
    NVStrings* strs = NVStrings::create_from_array(hstrs.data(), hstrs.size());

    thrust::device_vector<bool> results(hstrs.size(),false);
    strs->isupper(results.data().get());
    {
        bool expected[] = { false, false, false, true, false, false,
                            true, true, true, true, false, true, true,
                            true, false, true };
        for( int idx = 0; idx < (int) hstrs.size(); ++idx )
            EXPECT_EQ(results[idx],expected[idx]);
    }

    strs->islower(results.data().get());
    {
        bool expected[] = { false, true, false, false, true, false,
                            true, true, true, true, true, true, true,
                            true, true, true };
        for( int idx = 0; idx < (int) hstrs.size(); ++idx )
            EXPECT_EQ(results[idx],expected[idx]);
    }

    NVStrings::destroy(strs);
}


int main( int argc, char** argv )
{
    testing::InitGoogleTest(&argc,argv);
    return RUN_ALL_TESTS();
}