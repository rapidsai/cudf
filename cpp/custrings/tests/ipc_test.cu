#include <stdio.h>
#include <string>
#include <hip/hip_runtime.h>
#include <thrust/device_vector.h>
#include <thrust/execution_policy.h>
#include <thrust/for_each.h>
#include <thrust/count.h>

#include "nvstrings/NVStrings.h"
#include "nvstrings/NVCategory.h"
#include "nvstrings/ipc_transfer.h"

//
// cd ../build
// nvcc -w -std=c++11 --expt-extended-lambda -gencode arch=compute_70,code=sm_70 ../tests/ipc_test.cu -L. -lNVStrings -lNVCategory -o ipc_test --linker-options -rpath,.:
//

int strings_test( std::string& mode )
{
    NVStrings* strs = 0;
    if( mode.compare("client")==0 )
    {
        nvstrings_ipc_transfer ipc;
        FILE* fh = fopen("ipctx.bin","rb");
        fread(&ipc,1,sizeof(ipc),fh);
        fclose(fh);
        printf("%p %ld %ld\n", ipc.base_address, ipc.count, ipc.size);
        strs = NVStrings::create_from_ipc(ipc);
        strs->print();
        printf("%u strings in %ld bytes\n", strs->size(), strs->memsize() );
    }
    else
    {
        const char* hstrs[] = { "John Smith", "Joe Blow", "Jane Smith" };
        strs = NVStrings::create_from_array(hstrs,3);
        nvstrings_ipc_transfer ipc;
        strs->create_ipc_transfer(ipc);
        //printf("%p %ld %ld\n", ipc.base_address, ipc.count, ipc.size);
        strs->print();
        printf("%u strings in %ld bytes\n", strs->size(), strs->memsize() );
        FILE* fh = fopen("ipctx.bin","wb");
        fwrite((void*)&ipc,1,sizeof(ipc),fh);
        fclose(fh);
        printf("Server ready. Press enter to terminate.\n");
        std::cin.ignore();
        // just checking
        strs->print();
    }

    NVStrings::destroy(strs);
    return 0;
}

int category_test( std::string& mode )
{
    NVCategory* cat = 0;
    if( mode.compare("client")==0 )
    {
        nvcategory_ipc_transfer ipc;
        FILE* fh = fopen("ipctx.bin","rb");
        fread(&ipc,1,sizeof(ipc),fh);
        fclose(fh);
        cat = NVCategory::create_from_ipc(ipc);
        //printf("%p %p:%u %p:%u %p:%ld\n", ipc.base_address, ipc.strs, ipc.keys, ipc.vals, ipc.count, ipc.mem, ipc.size);
        NVStrings* strs = cat->get_keys();
        strs->print();
        NVStrings::destroy(strs);
    }
    else
    {
        const char* hstrs[] = { "John", "Jane", "John", "Jane", "Bob" };
        NVStrings* strs = NVStrings::create_from_array(hstrs,5);
        cat = NVCategory::create_from_strings(*strs);
        nvcategory_ipc_transfer ipc;
        cat->create_ipc_transfer(ipc);
        //printf("%p %p:%u %p:%u %p:%ld\n", ipc.base_address, ipc.strs, ipc.keys, ipc.vals, ipc.count, ipc.mem, ipc.size);
        NVStrings::destroy(strs);
        strs = cat->get_keys();
        strs->print();
        NVStrings::destroy(strs);

        FILE* fh = fopen("ipctx.bin","wb");
        fwrite((void*)&ipc,1,sizeof(ipc),fh);
        fclose(fh);
        printf("Server ready. Press enter to terminate.\n");
        std::cin.ignore();
    }

    NVCategory::destroy(cat);
    return 0;
}

int main( int argc, const char** argv )
{
    if( argc < 2 )
    {
        printf("require parameter: 'server' or values for pointers\n");
        return 0;
    }
    std::string mode = argv[1];
    //strings_test(mode);
    category_test(mode);
 }