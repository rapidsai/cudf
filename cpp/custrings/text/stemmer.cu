/*
* Copyright (c) 2018-2019, NVIDIA CORPORATION.  All rights reserved.
*
* Licensed under the Apache License, Version 2.0 (the "License");
* you may not use this file except in compliance with the License.
* You may obtain a copy of the License at
*
*     http://www.apache.org/licenses/LICENSE-2.0
*
* Unless required by applicable law or agreed to in writing, software
* distributed under the License is distributed on an "AS IS" BASIS,
* WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
* See the License for the specific language governing permissions and
* limitations under the License.
*/

#include <hip/hip_runtime.h>
#include <thrust/device_vector.h>
#include <thrust/for_each.h>
#include <rmm/rmm.h>
#include <rmm/thrust_rmm_allocator.h>
#include <utilities/error_utils.hpp>

#include "nvstrings/NVStrings.h"
#include "nvstrings/NVText.h"

#include "../custring_view.cuh"
#include "../util.h"

struct porter_stemmer_measure_fn
{
    custring_view_array d_strings;
    custring_view* d_vowels;
    Char y_char;
    unsigned int* d_results;

    __device__ bool is_consonant( custring_view* dstr, int index )
    {
        Char ch = dstr->at(index);
        if( d_vowels->find(ch) >= 0 )
            return false;
        if( (ch != y_char) || (index==0) )
            return true;
        ch = dstr->at(index-1);       // only if previous char
        return d_vowels->find(ch)>=0; // is not a consonant
    }

    __device__ void operator()(unsigned int idx)
    {
        custring_view* dstr = d_strings[idx];
        if( !dstr )
            return;
        unsigned int vcs = 0;
        bool vowel_run = !is_consonant(dstr,0);
        for( auto itr=dstr->begin(); itr!=dstr->end(); itr++ )
        {
            if( is_consonant(dstr,itr.position()) )
            {
                if( vowel_run )
                    vcs++;
                vowel_run = false;
            }
            else
                vowel_run = true;
        }
        d_results[idx] = vcs;
    }
};

unsigned int NVText::porter_stemmer_measure(NVStrings& strs, const char* vowels, const char* y_char, unsigned int* results, bool bdevmem )
{
    unsigned int count = strs.size();
    if( count==0 )
        return 0; // nothing to do
    auto execpol = rmm::exec_policy(0);
    // setup results vector
    unsigned int* d_results = results;
    if( !bdevmem )
        d_results = device_alloc<unsigned int>(count,0);
    if( vowels==nullptr )
        vowels = "aeiou";
    custring_view* d_vowels = custring_from_host(vowels);
    if( y_char==nullptr )
        y_char = "y";
    Char char_y;
    custring_view::char_to_Char(y_char,char_y);

    // get the string pointers
    rmm::device_vector<custring_view*> strings(count,nullptr);
    custring_view** d_strings = strings.data().get();
    strs.create_custring_index(d_strings);

    // do the measure
    thrust::for_each_n(execpol->on(0), thrust::make_counting_iterator<unsigned int>(0), count,
        porter_stemmer_measure_fn{d_strings,d_vowels,char_y,d_results});

    // done
    if( !bdevmem )
    {
        CUDA_TRY( hipMemcpyAsync(results,d_results,count*sizeof(unsigned int),hipMemcpyDeviceToHost))
        RMM_FREE(d_results,0);
    }
    RMM_FREE(d_vowels,0);
    return 0;
}