#include <benchmark/benchmark.h>

static void BM_StringCreation(benchmark::State& state) {
  for (auto _ : state)
    std::string empty_string;
}
// Register the function as a benchmark
BENCHMARK(BM_StringCreation);

// Define another benchmark
static void BM_StringCopy(benchmark::State& state) {
  std::string x = "hello";
  for (auto _ : state)
    std::string copy(x);
}
BENCHMARK(BM_StringCopy);

static void BM_StringCompare(benchmark::State& state) {
  std::string s1(state.range(0), '-');
  std::string s2(state.range(0), '-');
  for (auto _ : state) {
    benchmark::DoNotOptimize(s1.compare(s2));
  }
  state.SetComplexityN(state.range(0));
}
BENCHMARK(BM_StringCompare)
    ->RangeMultiplier(2)->Range(1<<10, 1<<18)->Complexity(benchmark::oN);

template <class Q> 
void BM_Sequential(benchmark::State& state) {
  Q q;
  typename Q::value_type v(0);
  for (auto _ : state) {
    for (int i = state.range(0); i--; )
      q.push_back(v);
  }
  // actually messages, not bytes:
  state.SetBytesProcessed(
      static_cast<int64_t>(state.iterations())*state.range(0));
}
BENCHMARK_TEMPLATE(BM_Sequential, std::vector<int>)->Range(1<<0, 1<<10);

BENCHMARK_MAIN();
