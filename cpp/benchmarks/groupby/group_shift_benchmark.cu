#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2021, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */
#include <benchmarks/fixture/benchmark_fixture.hpp>
#include <benchmarks/synchronization/synchronization.hpp>

#include <cudf/detail/iterator.cuh>
#include <cudf/groupby.hpp>
#include <cudf/scalar/scalar_factories.hpp>
#include <cudf/table/table_view.hpp>
#include <cudf/types.hpp>

#include <cudf_test/column_wrapper.hpp>

#include <benchmark/benchmark.h>

#include <random>

class Groupby : public cudf::benchmark {
};

// TODO: put it in a struct so `uniform` can be remade with different min, max
template <typename T>
T random_int(T min, T max)
{
  static unsigned seed = 13377331;
  static std::mt19937 engine{seed};
  static std::uniform_int_distribution<T> uniform{min, max};

  return uniform(engine);
}

void BM_group_shift(benchmark::State& state)
{
  using wrapper = cudf::test::fixed_width_column_wrapper<int64_t>;

  const cudf::size_type column_size{(cudf::size_type)state.range(0)};
  const int num_groups = 100;

  auto data_it = cudf::detail::make_counting_transform_iterator(
    0, [](cudf::size_type row) { return random_int(0, num_groups); });

  wrapper keys(data_it, data_it + column_size);
  wrapper vals(data_it, data_it + column_size);

  cudf::groupby::groupby gb_obj(cudf::table_view({keys}));

  cudf::size_type offset =
    static_cast<cudf::size_type>(column_size / float(num_groups) * 0.5);  // forward shift half way
  // null fill value
  auto fill_value = cudf::make_default_constructed_scalar(cudf::data_type(cudf::type_id::INT64));
  // non null fill value
  // auto fill_value = cudf::make_fixed_width_scalar(static_cast<int64_t>(42));

  for (auto _ : state) {
    cuda_event_timer timer(state, true);
    auto result = gb_obj.shift(vals, offset, *fill_value);
  }
}

BENCHMARK_DEFINE_F(Groupby, Shift)(::benchmark::State& state) { BM_group_shift(state); }

BENCHMARK_REGISTER_F(Groupby, Shift)
  ->Arg(1000000)
  ->Arg(10000000)
  ->Arg(100000000)
  ->UseManualTime()
  ->Unit(benchmark::kMillisecond);
