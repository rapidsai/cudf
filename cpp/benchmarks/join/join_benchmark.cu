/*
 * Copyright (c) 2019, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <benchmark/benchmark.h>
#include <cudf/cudf.h>
#include <cudf/legacy/join.hpp>
#include <cudf/utilities/error.hpp>
#include <fixture/benchmark_fixture.hpp>
#include <synchronization/synchronization.hpp>
#include <tests/utilities/legacy/column_wrapper.cuh>
#include <vector>

#include "generate_input_tables.cuh"

template <typename key_type, typename payload_type>
class Join : public cudf::benchmark {};

template <typename key_type, typename payload_type>
static void BM_join(benchmark::State &state) {
  const cudf::size_type build_table_size{(cudf::size_type)state.range(0)};
  const cudf::size_type probe_table_size{(cudf::size_type)state.range(1)};
  const cudf::size_type rand_max_val{build_table_size * 2};
  const double selectivity             = 0.3;
  const bool is_build_table_key_unique = true;

  // Generate build and probe tables

  cudf::test::column_wrapper<key_type> build_key_column(build_table_size);
  cudf::test::column_wrapper<key_type> probe_key_column(probe_table_size);

  generate_input_tables<key_type, cudf::size_type>((key_type *)build_key_column.get()->data,
                                                   build_table_size,
                                                   (key_type *)probe_key_column.get()->data,
                                                   probe_table_size,
                                                   selectivity,
                                                   rand_max_val,
                                                   is_build_table_key_unique);

  cudf::test::column_wrapper<payload_type> build_payload_column(
    build_table_size, [](cudf::size_type row_index) { return row_index; });

  cudf::test::column_wrapper<payload_type> probe_payload_column(
    probe_table_size, [](cudf::size_type row_index) { return row_index; });

  CHECK_CUDA(0);

  cudf::table build_table{build_key_column.get(), build_payload_column.get()};
  cudf::table probe_table{probe_key_column.get(), probe_payload_column.get()};

  // Setup join parameters and result table

  gdf_context ctxt = {
    0,                    // input data is not sorted
    gdf_method::GDF_HASH  // hash based join
  };

  std::vector<cudf::size_type> columns_to_join = {0};

  // Benchmark the inner join operation

  for (auto _ : state) {
    cuda_event_timer raii(state, true, 0);

    cudf::table result = cudf::inner_join(
      probe_table, build_table, columns_to_join, columns_to_join, {{0, 0}}, nullptr, &ctxt);

    result.destroy();
  }
}

#define JOIN_BENCHMARK_DEFINE(name, key_type, payload_type)       \
  BENCHMARK_TEMPLATE_DEFINE_F(Join, name, key_type, payload_type) \
  (::benchmark::State & st) { BM_join<key_type, payload_type>(st); }

JOIN_BENCHMARK_DEFINE(join_32bit, int32_t, int32_t);
JOIN_BENCHMARK_DEFINE(join_64bit, int64_t, int64_t);

BENCHMARK_REGISTER_F(Join, join_32bit)
  ->Unit(benchmark::kMillisecond)
  ->Args({100'000, 100'000})
  ->Args({100'000, 400'000})
  ->Args({100'000, 1'000'000})
  ->Args({10'000'000, 10'000'000})
  ->Args({10'000'000, 40'000'000})
  ->Args({10'000'000, 100'000'000})
  ->Args({100'000'000, 100'000'000})
  ->Args({80'000'000, 240'000'000})
  ->UseManualTime();

BENCHMARK_REGISTER_F(Join, join_64bit)
  ->Unit(benchmark::kMillisecond)
  ->Args({50'000'000, 50'000'000})
  ->Args({40'000'000, 120'000'000})
  ->UseManualTime();
