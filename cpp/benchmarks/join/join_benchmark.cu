/*
 * Copyright (c) 2019, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <benchmark/benchmark.h>
#include <vector>
#include <cudf/cudf.h>
#include <utilities/error_utils.hpp>
#include <tests/utilities/column_wrapper.cuh>

#include "generate_input_tables.cuh"
#include "../synchronization/synchronization.hpp"


template<typename key_type, typename payload_type>
static void join_benchmark(benchmark::State& state)
{
    const gdf_size_type build_table_size {(gdf_size_type) state.range(0)};
    const gdf_size_type probe_table_size {(gdf_size_type) state.range(1)};
    const gdf_size_type rand_max_val {build_table_size * 2};
    const double selectivity = 0.3;
    const bool is_build_table_key_unique = true;

    // Generate build and probe tables

    cudf::test::column_wrapper<key_type> build_key_column(build_table_size);
    cudf::test::column_wrapper<key_type> probe_key_column(probe_table_size);

    generate_input_tables<key_type, gdf_size_type>(
        (key_type *)build_key_column.get()->data, build_table_size,
        (key_type *)probe_key_column.get()->data, probe_table_size,
        selectivity, rand_max_val, is_build_table_key_unique
    );

    cudf::test::column_wrapper<payload_type> build_payload_column(
        build_table_size,
        [] (gdf_index_type row_index) {
            return row_index;
        }
    );

    cudf::test::column_wrapper<payload_type> probe_payload_column(
        probe_table_size,
        [] (gdf_index_type row_index) {
            return row_index;
        }
    );

    CHECK_STREAM(0);

    std::vector<gdf_column*> build_table {build_key_column.get(), build_payload_column.get()};
    std::vector<gdf_column*> probe_table {probe_key_column.get(), probe_payload_column.get()};

    // Setup join parameters and result table

    gdf_context ctxt = {
        0,                     // input data is not sorted
        gdf_method::GDF_HASH   // hash based join
    };

    int columns_to_join[] = {0};

    const int nresult_cols = build_table.size() + probe_table.size() - 1;

    std::vector<gdf_column> result_table(nresult_cols);
    std::vector<gdf_column *> col_ptrs(nresult_cols);

    for (size_t icol = 0; icol < result_table.size(); icol++) {
        col_ptrs[icol] = &result_table[icol];
    }

    // Benchmark the inner join operation

    for (auto _ : state) {
        cuda_event_timer raii(state, true, 0);

        CUDF_TRY(gdf_inner_join(
            probe_table.data(), 2, columns_to_join,
            build_table.data(), 2, columns_to_join,
            1, nresult_cols, col_ptrs.data(),
            nullptr, nullptr, &ctxt
        ));
    }

    // Cleanup

    for (auto & col_ptr : col_ptrs) {
        CUDF_TRY(gdf_column_free(col_ptr));
    }
}

BENCHMARK_TEMPLATE(join_benchmark, int32_t, int32_t)->Unit(benchmark::kMillisecond)
    ->Args({100'000, 100'000})
    ->Args({100'000, 400'000})
    ->Args({100'000, 1'000'000})
    ->Args({10'000'000, 10'000'000})
    ->Args({10'000'000, 40'000'000})
    ->Args({10'000'000, 100'000'000})
    ->Args({100'000'000, 100'000'000})
    ->Args({80'000'000, 240'000'000})
    ->UseManualTime();

BENCHMARK_TEMPLATE(join_benchmark, int64_t, int64_t)->Unit(benchmark::kMillisecond)
    ->Args({50'000'000, 50'000'000})
    ->Args({40'000'000, 120'000'000})
    ->UseManualTime();
