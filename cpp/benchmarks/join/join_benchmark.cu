/*
 * Copyright (c) 2019, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <benchmark/benchmark.h>

#include <thrust/iterator/counting_iterator.h>

#include <cudf/column/column_factories.hpp>
#include <cudf/join.hpp>
#include <cudf/table/table.hpp>
#include <cudf/table/table_view.hpp>
#include <cudf/utilities/error.hpp>
#include <cudf_test/column_wrapper.hpp>

#include <fixture/benchmark_fixture.hpp>
#include <synchronization/synchronization.hpp>

#include <vector>

#include "generate_input_tables.cuh"

template <typename key_type, typename payload_type>
class Join : public cudf::benchmark {
};

template <typename key_type, typename payload_type>
static void BM_join(benchmark::State &state)
{
  const cudf::size_type build_table_size{(cudf::size_type)state.range(0)};
  const cudf::size_type probe_table_size{(cudf::size_type)state.range(1)};
  const cudf::size_type rand_max_val{build_table_size * 2};
  const double selectivity             = 0.3;
  const bool is_build_table_key_unique = true;

  // Generate build and probe tables

  auto build_key_column =
    cudf::make_numeric_column(cudf::data_type(cudf::type_to_id<key_type>()), build_table_size);
  auto probe_key_column =
    cudf::make_numeric_column(cudf::data_type(cudf::type_to_id<key_type>()), probe_table_size);

  generate_input_tables<key_type, cudf::size_type>(
    build_key_column->mutable_view().data<key_type>(),
    build_table_size,
    probe_key_column->mutable_view().data<key_type>(),
    probe_table_size,
    selectivity,
    rand_max_val,
    is_build_table_key_unique);

  auto payload_data_it = thrust::make_counting_iterator(0);
  cudf::test::fixed_width_column_wrapper<payload_type> build_payload_column(
    payload_data_it, payload_data_it + build_table_size);

  cudf::test::fixed_width_column_wrapper<payload_type> probe_payload_column(
    payload_data_it, payload_data_it + probe_table_size);

  CHECK_CUDA(0);

  cudf::table_view build_table({build_key_column->view(), build_payload_column});
  cudf::table_view probe_table({probe_key_column->view(), probe_payload_column});

  // Setup join parameters and result table

  std::vector<cudf::size_type> columns_to_join = {0};

  // Benchmark the inner join operation

  for (auto _ : state) {
    cuda_event_timer raii(state, true, 0);

    auto result =
      cudf::inner_join(probe_table, build_table, columns_to_join, columns_to_join, {{0, 0}});
  }
}

#define JOIN_BENCHMARK_DEFINE(name, key_type, payload_type)       \
  BENCHMARK_TEMPLATE_DEFINE_F(Join, name, key_type, payload_type) \
  (::benchmark::State & st) { BM_join<key_type, payload_type>(st); }

JOIN_BENCHMARK_DEFINE(join_32bit, int32_t, int32_t);
JOIN_BENCHMARK_DEFINE(join_64bit, int64_t, int64_t);

BENCHMARK_REGISTER_F(Join, join_32bit)
  ->Unit(benchmark::kMillisecond)
  ->Args({100'000, 100'000})
  ->Args({100'000, 400'000})
  ->Args({100'000, 1'000'000})
  ->Args({10'000'000, 10'000'000})
  ->Args({10'000'000, 40'000'000})
  ->Args({10'000'000, 100'000'000})
  ->Args({100'000'000, 100'000'000})
  ->Args({80'000'000, 240'000'000})
  ->UseManualTime();

BENCHMARK_REGISTER_F(Join, join_64bit)
  ->Unit(benchmark::kMillisecond)
  ->Args({50'000'000, 50'000'000})
  ->Args({40'000'000, 120'000'000})
  ->UseManualTime();
