/*
 * Copyright (c) 2019-2022, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <benchmarks/join/join_common.hpp>

template <typename key_type, typename payload_type>
class ConditionalJoin : public cudf::benchmark {
};

// For compatibility with the shared logic for equality (hash) joins, all of
// the join lambdas defined by these macros accept a null_equality parameter
// but ignore it (don't forward it to the underlying join implementation)
// because conditional joins do not use this parameter.
#define CONDITIONAL_INNER_JOIN_BENCHMARK_DEFINE(name, key_type, payload_type, nullable) \
  BENCHMARK_TEMPLATE_DEFINE_F(ConditionalJoin, name, key_type, payload_type)            \
  (::benchmark::State & st)                                                             \
  {                                                                                     \
    auto join = [](cudf::table_view const& left,                                        \
                   cudf::table_view const& right,                                       \
                   cudf::ast::operation binary_pred,                                    \
                   cudf::null_equality compare_nulls) {                                 \
      return cudf::conditional_inner_join(left, right, binary_pred);                    \
    };                                                                                  \
    constexpr bool is_conditional = true;                                               \
    BM_join<key_type, payload_type, nullable, is_conditional>(st, join);                \
  }

CONDITIONAL_INNER_JOIN_BENCHMARK_DEFINE(conditional_inner_join_32bit, int32_t, int32_t, false);
CONDITIONAL_INNER_JOIN_BENCHMARK_DEFINE(conditional_inner_join_64bit, int64_t, int64_t, false);
CONDITIONAL_INNER_JOIN_BENCHMARK_DEFINE(conditional_inner_join_32bit_nulls, int32_t, int32_t, true);
CONDITIONAL_INNER_JOIN_BENCHMARK_DEFINE(conditional_inner_join_64bit_nulls, int64_t, int64_t, true);

#define CONDITIONAL_LEFT_JOIN_BENCHMARK_DEFINE(name, key_type, payload_type, nullable) \
  BENCHMARK_TEMPLATE_DEFINE_F(ConditionalJoin, name, key_type, payload_type)           \
  (::benchmark::State & st)                                                            \
  {                                                                                    \
    auto join = [](cudf::table_view const& left,                                       \
                   cudf::table_view const& right,                                      \
                   cudf::ast::operation binary_pred,                                   \
                   cudf::null_equality compare_nulls) {                                \
      return cudf::conditional_left_join(left, right, binary_pred);                    \
    };                                                                                 \
    constexpr bool is_conditional = true;                                              \
    BM_join<key_type, payload_type, nullable, is_conditional>(st, join);               \
  }

CONDITIONAL_LEFT_JOIN_BENCHMARK_DEFINE(conditional_left_join_32bit, int32_t, int32_t, false);
CONDITIONAL_LEFT_JOIN_BENCHMARK_DEFINE(conditional_left_join_64bit, int64_t, int64_t, false);
CONDITIONAL_LEFT_JOIN_BENCHMARK_DEFINE(conditional_left_join_32bit_nulls, int32_t, int32_t, true);
CONDITIONAL_LEFT_JOIN_BENCHMARK_DEFINE(conditional_left_join_64bit_nulls, int64_t, int64_t, true);

#define CONDITIONAL_FULL_JOIN_BENCHMARK_DEFINE(name, key_type, payload_type, nullable) \
  BENCHMARK_TEMPLATE_DEFINE_F(ConditionalJoin, name, key_type, payload_type)           \
  (::benchmark::State & st)                                                            \
  {                                                                                    \
    auto join = [](cudf::table_view const& left,                                       \
                   cudf::table_view const& right,                                      \
                   cudf::ast::operation binary_pred,                                   \
                   cudf::null_equality compare_nulls) {                                \
      return cudf::conditional_full_join(left, right, binary_pred);                    \
    };                                                                                 \
    constexpr bool is_conditional = true;                                              \
    BM_join<key_type, payload_type, nullable, is_conditional>(st, join);               \
  }

CONDITIONAL_FULL_JOIN_BENCHMARK_DEFINE(conditional_full_join_32bit, int32_t, int32_t, false);
CONDITIONAL_FULL_JOIN_BENCHMARK_DEFINE(conditional_full_join_64bit, int64_t, int64_t, false);
CONDITIONAL_FULL_JOIN_BENCHMARK_DEFINE(conditional_full_join_32bit_nulls, int32_t, int32_t, true);
CONDITIONAL_FULL_JOIN_BENCHMARK_DEFINE(conditional_full_join_64bit_nulls, int64_t, int64_t, true);

#define CONDITIONAL_LEFT_ANTI_JOIN_BENCHMARK_DEFINE(name, key_type, payload_type, nullable) \
  BENCHMARK_TEMPLATE_DEFINE_F(ConditionalJoin, name, key_type, payload_type)                \
  (::benchmark::State & st)                                                                 \
  {                                                                                         \
    auto join = [](cudf::table_view const& left,                                            \
                   cudf::table_view const& right,                                           \
                   cudf::ast::operation binary_pred,                                        \
                   cudf::null_equality compare_nulls) {                                     \
      return cudf::conditional_left_anti_join(left, right, binary_pred);                    \
    };                                                                                      \
    constexpr bool is_conditional = true;                                                   \
    BM_join<key_type, payload_type, nullable, is_conditional>(st, join);                    \
  }

CONDITIONAL_LEFT_ANTI_JOIN_BENCHMARK_DEFINE(conditional_left_anti_join_32bit,
                                            int32_t,
                                            int32_t,
                                            false);
CONDITIONAL_LEFT_ANTI_JOIN_BENCHMARK_DEFINE(conditional_left_anti_join_64bit,
                                            int64_t,
                                            int64_t,
                                            false);
CONDITIONAL_LEFT_ANTI_JOIN_BENCHMARK_DEFINE(conditional_left_anti_join_32bit_nulls,
                                            int32_t,
                                            int32_t,
                                            true);
CONDITIONAL_LEFT_ANTI_JOIN_BENCHMARK_DEFINE(conditional_left_anti_join_64bit_nulls,
                                            int64_t,
                                            int64_t,
                                            true);

#define CONDITIONAL_LEFT_SEMI_JOIN_BENCHMARK_DEFINE(name, key_type, payload_type, nullable) \
  BENCHMARK_TEMPLATE_DEFINE_F(ConditionalJoin, name, key_type, payload_type)                \
  (::benchmark::State & st)                                                                 \
  {                                                                                         \
    auto join = [](cudf::table_view const& left,                                            \
                   cudf::table_view const& right,                                           \
                   cudf::ast::operation binary_pred,                                        \
                   cudf::null_equality compare_nulls) {                                     \
      return cudf::conditional_left_semi_join(left, right, binary_pred);                    \
    };                                                                                      \
    constexpr bool is_conditional = true;                                                   \
    BM_join<key_type, payload_type, nullable, is_conditional>(st, join);                    \
  }

CONDITIONAL_LEFT_SEMI_JOIN_BENCHMARK_DEFINE(conditional_left_semi_join_32bit,
                                            int32_t,
                                            int32_t,
                                            false);
CONDITIONAL_LEFT_SEMI_JOIN_BENCHMARK_DEFINE(conditional_left_semi_join_64bit,
                                            int64_t,
                                            int64_t,
                                            false);
CONDITIONAL_LEFT_SEMI_JOIN_BENCHMARK_DEFINE(conditional_left_semi_join_32bit_nulls,
                                            int32_t,
                                            int32_t,
                                            true);
CONDITIONAL_LEFT_SEMI_JOIN_BENCHMARK_DEFINE(conditional_left_semi_join_64bit_nulls,
                                            int64_t,
                                            int64_t,
                                            true);

// inner join -----------------------------------------------------------------------
BENCHMARK_REGISTER_F(ConditionalJoin, conditional_inner_join_32bit)
  ->Unit(benchmark::kMillisecond)
  ->Args({100'000, 100'000})
  ->Args({100'000, 400'000})
  ->Args({400'000, 100'000})
  ->Args({100'000, 1'000'000})
  ->UseManualTime();

BENCHMARK_REGISTER_F(ConditionalJoin, conditional_inner_join_64bit)
  ->Unit(benchmark::kMillisecond)
  ->Args({100'000, 100'000})
  ->Args({100'000, 400'000})
  ->Args({400'000, 100'000})
  ->Args({100'000, 1'000'000})
  ->UseManualTime();

BENCHMARK_REGISTER_F(ConditionalJoin, conditional_inner_join_32bit_nulls)
  ->Unit(benchmark::kMillisecond)
  ->Args({100'000, 100'000})
  ->Args({100'000, 400'000})
  ->Args({400'000, 100'000})
  ->Args({100'000, 1'000'000})
  ->UseManualTime();

BENCHMARK_REGISTER_F(ConditionalJoin, conditional_inner_join_64bit_nulls)
  ->Unit(benchmark::kMillisecond)
  ->Args({100'000, 100'000})
  ->Args({100'000, 400'000})
  ->Args({400'000, 100'000})
  ->Args({100'000, 1'000'000})
  ->UseManualTime();

// left join -----------------------------------------------------------------------
BENCHMARK_REGISTER_F(ConditionalJoin, conditional_left_join_32bit)
  ->Unit(benchmark::kMillisecond)
  ->Args({100'000, 100'000})
  ->Args({100'000, 400'000})
  ->Args({100'000, 1'000'000})
  ->UseManualTime();

BENCHMARK_REGISTER_F(ConditionalJoin, conditional_left_join_64bit)
  ->Unit(benchmark::kMillisecond)
  ->Args({100'000, 100'000})
  ->Args({100'000, 400'000})
  ->Args({100'000, 1'000'000})
  ->UseManualTime();

BENCHMARK_REGISTER_F(ConditionalJoin, conditional_left_join_32bit_nulls)
  ->Unit(benchmark::kMillisecond)
  ->Args({100'000, 100'000})
  ->Args({100'000, 400'000})
  ->Args({100'000, 1'000'000})
  ->UseManualTime();

BENCHMARK_REGISTER_F(ConditionalJoin, conditional_left_join_64bit_nulls)
  ->Unit(benchmark::kMillisecond)
  ->Args({100'000, 100'000})
  ->Args({100'000, 400'000})
  ->Args({100'000, 1'000'000})
  ->UseManualTime();

// full join -----------------------------------------------------------------------
BENCHMARK_REGISTER_F(ConditionalJoin, conditional_full_join_32bit)
  ->Unit(benchmark::kMillisecond)
  ->Args({100'000, 100'000})
  ->Args({100'000, 400'000})
  ->Args({100'000, 1'000'000})
  ->UseManualTime();

BENCHMARK_REGISTER_F(ConditionalJoin, conditional_full_join_64bit)
  ->Unit(benchmark::kMillisecond)
  ->Args({100'000, 100'000})
  ->Args({100'000, 400'000})
  ->Args({100'000, 1'000'000})
  ->UseManualTime();

BENCHMARK_REGISTER_F(ConditionalJoin, conditional_full_join_32bit_nulls)
  ->Unit(benchmark::kMillisecond)
  ->Args({100'000, 100'000})
  ->Args({100'000, 400'000})
  ->Args({100'000, 1'000'000})
  ->UseManualTime();

BENCHMARK_REGISTER_F(ConditionalJoin, conditional_full_join_64bit_nulls)
  ->Unit(benchmark::kMillisecond)
  ->Args({100'000, 100'000})
  ->Args({100'000, 400'000})
  ->Args({100'000, 1'000'000})
  ->UseManualTime();

// left anti-join -------------------------------------------------------------
BENCHMARK_REGISTER_F(ConditionalJoin, conditional_left_anti_join_32bit)
  ->Unit(benchmark::kMillisecond)
  ->Args({100'000, 100'000})
  ->Args({100'000, 400'000})
  ->Args({100'000, 1'000'000})
  ->UseManualTime();

BENCHMARK_REGISTER_F(ConditionalJoin, conditional_left_anti_join_64bit)
  ->Unit(benchmark::kMillisecond)
  ->Args({100'000, 100'000})
  ->Args({100'000, 400'000})
  ->Args({100'000, 1'000'000})
  ->UseManualTime();

BENCHMARK_REGISTER_F(ConditionalJoin, conditional_left_anti_join_32bit_nulls)
  ->Unit(benchmark::kMillisecond)
  ->Args({100'000, 100'000})
  ->Args({100'000, 400'000})
  ->Args({100'000, 1'000'000})
  ->UseManualTime();

BENCHMARK_REGISTER_F(ConditionalJoin, conditional_left_anti_join_64bit_nulls)
  ->Unit(benchmark::kMillisecond)
  ->Args({100'000, 100'000})
  ->Args({100'000, 400'000})
  ->Args({100'000, 1'000'000})
  ->UseManualTime();

// left semi-join -------------------------------------------------------------
BENCHMARK_REGISTER_F(ConditionalJoin, conditional_left_semi_join_32bit)
  ->Unit(benchmark::kMillisecond)
  ->Args({100'000, 100'000})
  ->Args({100'000, 400'000})
  ->Args({100'000, 1'000'000})
  ->UseManualTime();

BENCHMARK_REGISTER_F(ConditionalJoin, conditional_left_semi_join_64bit)
  ->Unit(benchmark::kMillisecond)
  ->Args({100'000, 100'000})
  ->Args({100'000, 400'000})
  ->Args({100'000, 1'000'000})
  ->UseManualTime();

BENCHMARK_REGISTER_F(ConditionalJoin, conditional_left_semi_join_32bit_nulls)
  ->Unit(benchmark::kMillisecond)
  ->Args({100'000, 100'000})
  ->Args({100'000, 400'000})
  ->Args({100'000, 1'000'000})
  ->UseManualTime();

BENCHMARK_REGISTER_F(ConditionalJoin, conditional_left_semi_join_64bit_nulls)
  ->Unit(benchmark::kMillisecond)
  ->Args({100'000, 100'000})
  ->Args({100'000, 400'000})
  ->Args({100'000, 1'000'000})
  ->UseManualTime();
