/*
 * Copyright (c) 2019-2022, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <benchmarks/fixture/rmm_pool_raii.hpp>
#include <benchmarks/join/join_common.hpp>

void skip_helper(nvbench::state& state)
{
  auto const build_table_size = state.get_int64("Build Table Size");
  auto const probe_table_size = state.get_int64("Probe Table Size");

  if (build_table_size > probe_table_size) {
    state.skip("Large build tables are skipped.");
    return;
  }

  if (build_table_size * 100 <= probe_table_size) {
    state.skip("Large probe tables are skipped.");
    return;
  }
}

template <typename key_type, typename payload_type, bool Nullable>
void nvbench_inner_join(nvbench::state& state,
                        nvbench::type_list<key_type, payload_type, nvbench::enum_type<Nullable>>)
{
  skip_helper(state);

  // TODO: to be replaced by nvbench fixture once it's ready
  cudf::rmm_pool_raii pool_raii;

  auto join = [](cudf::table_view const& left_input,
                 cudf::table_view const& right_input,
                 cudf::null_equality compare_nulls,
                 rmm::cuda_stream_view stream) {
    cudf::hash_join hj_obj(left_input, compare_nulls, stream);
    return hj_obj.inner_join(right_input, std::nullopt, stream);
  };

  BM_join<key_type, payload_type, Nullable>(state, join);
}

template <typename key_type, typename payload_type, bool Nullable>
void nvbench_left_join(nvbench::state& state,
                       nvbench::type_list<key_type, payload_type, nvbench::enum_type<Nullable>>)
{
  skip_helper(state);

  // TODO: to be replaced by nvbench fixture once it's ready
  cudf::rmm_pool_raii pool_raii;

  auto join = [](cudf::table_view const& left_input,
                 cudf::table_view const& right_input,
                 cudf::null_equality compare_nulls,
                 rmm::cuda_stream_view stream) {
    cudf::hash_join hj_obj(left_input, compare_nulls, stream);
    return hj_obj.left_join(right_input, std::nullopt, stream);
  };

  BM_join<key_type, payload_type, Nullable>(state, join);
}

template <typename key_type, typename payload_type, bool Nullable>
void nvbench_full_join(nvbench::state& state,
                       nvbench::type_list<key_type, payload_type, nvbench::enum_type<Nullable>>)
{
  skip_helper(state);

  // TODO: to be replaced by nvbench fixture once it's ready
  cudf::rmm_pool_raii pool_raii;

  auto join = [](cudf::table_view const& left_input,
                 cudf::table_view const& right_input,
                 cudf::null_equality compare_nulls,
                 rmm::cuda_stream_view stream) {
    cudf::hash_join hj_obj(left_input, compare_nulls, stream);
    return hj_obj.full_join(right_input, std::nullopt, stream);
  };

  BM_join<key_type, payload_type, Nullable>(state, join);
}

// inner join -----------------------------------------------------------------------
NVBENCH_BENCH_TYPES(nvbench_inner_join,
                    NVBENCH_TYPE_AXES(nvbench::type_list<nvbench::int32_t>,
                                      nvbench::type_list<nvbench::int32_t>,
                                      nvbench::enum_type_list<false>))
  .set_name("inner_join_32bit")
  .set_type_axes_names({"Key Type", "Payload Type", "Nullable"})
  .add_int64_axis("Build Table Size", {100'000, 10'000'000, 80'000'000, 100'000'000})
  .add_int64_axis("Probe Table Size",
                  {100'000, 400'000, 10'000'000, 40'000'000, 100'000'000, 240'000'000});

NVBENCH_BENCH_TYPES(nvbench_inner_join,
                    NVBENCH_TYPE_AXES(nvbench::type_list<nvbench::int64_t>,
                                      nvbench::type_list<nvbench::int64_t>,
                                      nvbench::enum_type_list<false>))
  .set_name("inner_join_64bit")
  .set_type_axes_names({"Key Type", "Payload Type", "Nullable"})
  .add_int64_axis("Build Table Size", {40'000'000, 50'000'000})
  .add_int64_axis("Probe Table Size", {50'000'000, 120'000'000});

NVBENCH_BENCH_TYPES(nvbench_inner_join,
                    NVBENCH_TYPE_AXES(nvbench::type_list<nvbench::int32_t>,
                                      nvbench::type_list<nvbench::int32_t>,
                                      nvbench::enum_type_list<true>))
  .set_name("inner_join_32bit_nulls")
  .set_type_axes_names({"Key Type", "Payload Type", "Nullable"})
  .add_int64_axis("Build Table Size", {100'000, 10'000'000, 80'000'000, 100'000'000})
  .add_int64_axis("Probe Table Size",
                  {100'000, 400'000, 10'000'000, 40'000'000, 100'000'000, 240'000'000});

NVBENCH_BENCH_TYPES(nvbench_inner_join,
                    NVBENCH_TYPE_AXES(nvbench::type_list<nvbench::int64_t>,
                                      nvbench::type_list<nvbench::int64_t>,
                                      nvbench::enum_type_list<true>))
  .set_name("inner_join_64bit_nulls")
  .set_type_axes_names({"Key Type", "Payload Type", "Nullable"})
  .add_int64_axis("Build Table Size", {40'000'000, 50'000'000})
  .add_int64_axis("Probe Table Size", {50'000'000, 120'000'000});

// left join ------------------------------------------------------------------------
NVBENCH_BENCH_TYPES(nvbench_left_join,
                    NVBENCH_TYPE_AXES(nvbench::type_list<nvbench::int32_t>,
                                      nvbench::type_list<nvbench::int32_t>,
                                      nvbench::enum_type_list<false>))
  .set_name("left_join_32bit")
  .set_type_axes_names({"Key Type", "Payload Type", "Nullable"})
  .add_int64_axis("Build Table Size", {100'000, 10'000'000, 80'000'000, 100'000'000})
  .add_int64_axis("Probe Table Size",
                  {100'000, 400'000, 10'000'000, 40'000'000, 100'000'000, 240'000'000});

NVBENCH_BENCH_TYPES(nvbench_left_join,
                    NVBENCH_TYPE_AXES(nvbench::type_list<nvbench::int64_t>,
                                      nvbench::type_list<nvbench::int64_t>,
                                      nvbench::enum_type_list<false>))
  .set_name("left_join_64bit")
  .set_type_axes_names({"Key Type", "Payload Type", "Nullable"})
  .add_int64_axis("Build Table Size", {40'000'000, 50'000'000})
  .add_int64_axis("Probe Table Size", {50'000'000, 120'000'000});

NVBENCH_BENCH_TYPES(nvbench_left_join,
                    NVBENCH_TYPE_AXES(nvbench::type_list<nvbench::int32_t>,
                                      nvbench::type_list<nvbench::int32_t>,
                                      nvbench::enum_type_list<true>))
  .set_name("left_join_32bit_nulls")
  .set_type_axes_names({"Key Type", "Payload Type", "Nullable"})
  .add_int64_axis("Build Table Size", {100'000, 10'000'000, 80'000'000, 100'000'000})
  .add_int64_axis("Probe Table Size",
                  {100'000, 400'000, 10'000'000, 40'000'000, 100'000'000, 240'000'000});

NVBENCH_BENCH_TYPES(nvbench_left_join,
                    NVBENCH_TYPE_AXES(nvbench::type_list<nvbench::int64_t>,
                                      nvbench::type_list<nvbench::int64_t>,
                                      nvbench::enum_type_list<true>))
  .set_name("left_join_64bit_nulls")
  .set_type_axes_names({"Key Type", "Payload Type", "Nullable"})
  .add_int64_axis("Build Table Size", {40'000'000, 50'000'000})
  .add_int64_axis("Probe Table Size", {50'000'000, 120'000'000});

// full join ------------------------------------------------------------------------
NVBENCH_BENCH_TYPES(nvbench_full_join,
                    NVBENCH_TYPE_AXES(nvbench::type_list<nvbench::int32_t>,
                                      nvbench::type_list<nvbench::int32_t>,
                                      nvbench::enum_type_list<false>))
  .set_name("full_join_32bit")
  .set_type_axes_names({"Key Type", "Payload Type", "Nullable"})
  .add_int64_axis("Build Table Size", {100'000, 10'000'000, 80'000'000, 100'000'000})
  .add_int64_axis("Probe Table Size",
                  {100'000, 400'000, 10'000'000, 40'000'000, 100'000'000, 240'000'000});

NVBENCH_BENCH_TYPES(nvbench_full_join,
                    NVBENCH_TYPE_AXES(nvbench::type_list<nvbench::int64_t>,
                                      nvbench::type_list<nvbench::int64_t>,
                                      nvbench::enum_type_list<false>))
  .set_name("full_join_64bit")
  .set_type_axes_names({"Key Type", "Payload Type", "Nullable"})
  .add_int64_axis("Build Table Size", {40'000'000, 50'000'000})
  .add_int64_axis("Probe Table Size", {50'000'000, 120'000'000});

NVBENCH_BENCH_TYPES(nvbench_full_join,
                    NVBENCH_TYPE_AXES(nvbench::type_list<nvbench::int32_t>,
                                      nvbench::type_list<nvbench::int32_t>,
                                      nvbench::enum_type_list<true>))
  .set_name("full_join_32bit_nulls")
  .set_type_axes_names({"Key Type", "Payload Type", "Nullable"})
  .add_int64_axis("Build Table Size", {100'000, 10'000'000, 80'000'000, 100'000'000})
  .add_int64_axis("Probe Table Size",
                  {100'000, 400'000, 10'000'000, 40'000'000, 100'000'000, 240'000'000});

NVBENCH_BENCH_TYPES(nvbench_full_join,
                    NVBENCH_TYPE_AXES(nvbench::type_list<nvbench::int64_t>,
                                      nvbench::type_list<nvbench::int64_t>,
                                      nvbench::enum_type_list<true>))
  .set_name("full_join_64bit_nulls")
  .set_type_axes_names({"Key Type", "Payload Type", "Nullable"})
  .add_int64_axis("Build Table Size", {40'000'000, 50'000'000})
  .add_int64_axis("Probe Table Size", {50'000'000, 120'000'000});
