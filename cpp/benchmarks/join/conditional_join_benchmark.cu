/*
 * Copyright (c) 2019-2021, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <benchmark/benchmark.h>

#include <thrust/iterator/counting_iterator.h>

#include <cudf/ast/nodes.hpp>
#include <cudf/ast/operators.hpp>
#include <cudf/column/column_factories.hpp>
#include <cudf/join.hpp>
#include <cudf/table/table.hpp>
#include <cudf/table/table_view.hpp>
#include <cudf/utilities/error.hpp>
#include <cudf_test/base_fixture.hpp>
#include <cudf_test/column_wrapper.hpp>

#include <fixture/benchmark_fixture.hpp>
#include <synchronization/synchronization.hpp>

#include <vector>

#include "generate_input_tables.cuh"

template <typename key_type, typename payload_type>
class ConditionalJoin : public cudf::benchmark {
};

template <typename key_type, typename payload_type, bool Nullable, typename Join>
static void BM_join(benchmark::State& state, Join JoinFunc)
{
  const cudf::size_type build_table_size{(cudf::size_type)state.range(0)};
  const cudf::size_type probe_table_size{(cudf::size_type)state.range(1)};
  const cudf::size_type rand_max_val{build_table_size * 2};
  const double selectivity             = 0.3;
  const bool is_build_table_key_unique = true;

  // Generate build and probe tables
  cudf::test::UniformRandomGenerator<cudf::size_type> rand_gen(0, build_table_size);
  auto build_random_null_mask = [&rand_gen](int size) {
    if (Nullable) {
      // roughly 25% nulls
      auto validity = thrust::make_transform_iterator(
        thrust::make_counting_iterator(0),
        [&rand_gen](auto i) { return (rand_gen.generate() & 3) == 0; });
      return cudf::test::detail::make_null_mask(validity, validity + size);
    } else {
      return cudf::create_null_mask(size, cudf::mask_state::UNINITIALIZED);
    }
  };

  std::unique_ptr<cudf::column> build_key_column = [&]() {
    return Nullable ? cudf::make_numeric_column(cudf::data_type(cudf::type_to_id<key_type>()),
                                                build_table_size,
                                                build_random_null_mask(build_table_size))
                    : cudf::make_numeric_column(cudf::data_type(cudf::type_to_id<key_type>()),
                                                build_table_size);
  }();
  std::unique_ptr<cudf::column> probe_key_column = [&]() {
    return Nullable ? cudf::make_numeric_column(cudf::data_type(cudf::type_to_id<key_type>()),
                                                probe_table_size,
                                                build_random_null_mask(probe_table_size))
                    : cudf::make_numeric_column(cudf::data_type(cudf::type_to_id<key_type>()),
                                                probe_table_size);
  }();

  generate_input_tables<key_type, cudf::size_type>(
    build_key_column->mutable_view().data<key_type>(),
    build_table_size,
    probe_key_column->mutable_view().data<key_type>(),
    probe_table_size,
    selectivity,
    rand_max_val,
    is_build_table_key_unique);

  auto payload_data_it = thrust::make_counting_iterator(0);
  cudf::test::fixed_width_column_wrapper<payload_type> build_payload_column(
    payload_data_it, payload_data_it + build_table_size);

  cudf::test::fixed_width_column_wrapper<payload_type> probe_payload_column(
    payload_data_it, payload_data_it + probe_table_size);

  CHECK_CUDA(0);

  cudf::table_view build_table({build_key_column->view(), build_payload_column});
  cudf::table_view probe_table({probe_key_column->view(), probe_payload_column});

  // Benchmark the inner join operation

  for (auto _ : state) {
    cuda_event_timer raii(state, true, rmm::cuda_stream_default);

    // Common column references.
    const auto col_ref_left_0  = cudf::ast::column_reference(0);
    const auto col_ref_right_0 = cudf::ast::column_reference(0, cudf::ast::table_reference::RIGHT);
    auto left_zero_eq_right_zero =
      cudf::ast::expression(cudf::ast::ast_operator::EQUAL, col_ref_left_0, col_ref_right_0);

    auto result =
      JoinFunc(probe_table, build_table, left_zero_eq_right_zero, cudf::null_equality::UNEQUAL);
  }
}

#define CONDITIONAL_INNER_JOIN_BENCHMARK_DEFINE(name, key_type, payload_type, nullable) \
  BENCHMARK_TEMPLATE_DEFINE_F(ConditionalJoin, name, key_type, payload_type)            \
  (::benchmark::State & st)                                                             \
  {                                                                                     \
    auto join = [](cudf::table_view const& left,                                        \
                   cudf::table_view const& right,                                       \
                   cudf::ast::expression binary_pred,                                   \
                   cudf::null_equality compare_nulls) {                                 \
      return cudf::conditional_inner_join(left, right, binary_pred, compare_nulls);     \
    };                                                                                  \
    BM_join<key_type, payload_type, nullable>(st, join);                                \
  }

CONDITIONAL_INNER_JOIN_BENCHMARK_DEFINE(conditional_inner_join_32bit, int32_t, int32_t, false);
CONDITIONAL_INNER_JOIN_BENCHMARK_DEFINE(conditional_inner_join_64bit, int64_t, int64_t, false);
CONDITIONAL_INNER_JOIN_BENCHMARK_DEFINE(conditional_inner_join_32bit_nulls, int32_t, int32_t, true);
CONDITIONAL_INNER_JOIN_BENCHMARK_DEFINE(conditional_inner_join_64bit_nulls, int64_t, int64_t, true);

#define CONDITIONAL_LEFT_JOIN_BENCHMARK_DEFINE(name, key_type, payload_type, nullable) \
  BENCHMARK_TEMPLATE_DEFINE_F(ConditionalJoin, name, key_type, payload_type)           \
  (::benchmark::State & st)                                                            \
  {                                                                                    \
    auto join = [](cudf::table_view const& left,                                       \
                   cudf::table_view const& right,                                      \
                   cudf::ast::expression binary_pred,                                  \
                   cudf::null_equality compare_nulls) {                                \
      return cudf::conditional_left_join(left, right, binary_pred, compare_nulls);     \
    };                                                                                 \
    BM_join<key_type, payload_type, nullable>(st, join);                               \
  }

CONDITIONAL_LEFT_JOIN_BENCHMARK_DEFINE(conditional_left_join_32bit, int32_t, int32_t, false);
CONDITIONAL_LEFT_JOIN_BENCHMARK_DEFINE(conditional_left_join_64bit, int64_t, int64_t, false);
CONDITIONAL_LEFT_JOIN_BENCHMARK_DEFINE(conditional_left_join_32bit_nulls, int32_t, int32_t, true);
CONDITIONAL_LEFT_JOIN_BENCHMARK_DEFINE(conditional_left_join_64bit_nulls, int64_t, int64_t, true);

#define CONDITIONAL_FULL_JOIN_BENCHMARK_DEFINE(name, key_type, payload_type, nullable) \
  BENCHMARK_TEMPLATE_DEFINE_F(ConditionalJoin, name, key_type, payload_type)           \
  (::benchmark::State & st)                                                            \
  {                                                                                    \
    auto join = [](cudf::table_view const& left,                                       \
                   cudf::table_view const& right,                                      \
                   cudf::ast::expression binary_pred,                                  \
                   cudf::null_equality compare_nulls) {                                \
      return cudf::conditional_inner_join(left, right, binary_pred, compare_nulls);    \
    };                                                                                 \
    BM_join<key_type, payload_type, nullable>(st, join);                               \
  }

CONDITIONAL_FULL_JOIN_BENCHMARK_DEFINE(conditional_full_join_32bit, int32_t, int32_t, false);
CONDITIONAL_FULL_JOIN_BENCHMARK_DEFINE(conditional_full_join_64bit, int64_t, int64_t, false);
CONDITIONAL_FULL_JOIN_BENCHMARK_DEFINE(conditional_full_join_32bit_nulls, int32_t, int32_t, true);
CONDITIONAL_FULL_JOIN_BENCHMARK_DEFINE(conditional_full_join_64bit_nulls, int64_t, int64_t, true);

#define CONDITIONAL_LEFT_ANTI_JOIN_BENCHMARK_DEFINE(name, key_type, payload_type, nullable) \
  BENCHMARK_TEMPLATE_DEFINE_F(ConditionalJoin, name, key_type, payload_type)                \
  (::benchmark::State & st)                                                                 \
  {                                                                                         \
    auto join = [](cudf::table_view const& left,                                            \
                   cudf::table_view const& right,                                           \
                   cudf::ast::expression binary_pred,                                       \
                   cudf::null_equality compare_nulls) {                                     \
      return cudf::conditional_left_anti_join(left, right, binary_pred, compare_nulls);     \
    };                                                                                      \
    BM_join<key_type, payload_type, nullable>(st, join);                                    \
  }

CONDITIONAL_LEFT_ANTI_JOIN_BENCHMARK_DEFINE(conditional_left_anti_join_32bit,
                                            int32_t,
                                            int32_t,
                                            false);
CONDITIONAL_LEFT_ANTI_JOIN_BENCHMARK_DEFINE(conditional_left_anti_join_64bit,
                                            int64_t,
                                            int64_t,
                                            false);
CONDITIONAL_LEFT_ANTI_JOIN_BENCHMARK_DEFINE(conditional_left_anti_join_32bit_nulls,
                                            int32_t,
                                            int32_t,
                                            true);
CONDITIONAL_LEFT_ANTI_JOIN_BENCHMARK_DEFINE(conditional_left_anti_join_64bit_nulls,
                                            int64_t,
                                            int64_t,
                                            true);

#define CONDITIONAL_LEFT_SEMI_JOIN_BENCHMARK_DEFINE(name, key_type, payload_type, nullable) \
  BENCHMARK_TEMPLATE_DEFINE_F(ConditionalJoin, name, key_type, payload_type)                \
  (::benchmark::State & st)                                                                 \
  {                                                                                         \
    auto join = [](cudf::table_view const& left,                                            \
                   cudf::table_view const& right,                                           \
                   cudf::ast::expression binary_pred,                                       \
                   cudf::null_equality compare_nulls) {                                     \
      return cudf::conditional_left_semi_join(left, right, binary_pred, compare_nulls);     \
    };                                                                                      \
    BM_join<key_type, payload_type, nullable>(st, join);                                    \
  }

CONDITIONAL_LEFT_SEMI_JOIN_BENCHMARK_DEFINE(conditional_left_semi_join_32bit,
                                            int32_t,
                                            int32_t,
                                            false);
CONDITIONAL_LEFT_SEMI_JOIN_BENCHMARK_DEFINE(conditional_left_semi_join_64bit,
                                            int64_t,
                                            int64_t,
                                            false);
CONDITIONAL_LEFT_SEMI_JOIN_BENCHMARK_DEFINE(conditional_left_semi_join_32bit_nulls,
                                            int32_t,
                                            int32_t,
                                            true);
CONDITIONAL_LEFT_SEMI_JOIN_BENCHMARK_DEFINE(conditional_left_semi_join_64bit_nulls,
                                            int64_t,
                                            int64_t,
                                            true);

// inner join -----------------------------------------------------------------------
BENCHMARK_REGISTER_F(ConditionalJoin, conditional_inner_join_32bit)
  ->Unit(benchmark::kMillisecond)
  ->Args({100'000, 100'000})
  ->Args({100'000, 400'000})
  ->Args({100'000, 1'000'000})
  // TODO: The below benchmark is slow, but can be useful to validate that the
  // code works for large data sets. This benchmark was used to compare to the
  // otherwise equivalent nullable benchmark below, which has memory errors for
  // sufficiently large data sets.
  //->Args({1'000'000, 1'000'000})
  ->UseManualTime();

BENCHMARK_REGISTER_F(ConditionalJoin, conditional_inner_join_64bit)
  ->Unit(benchmark::kMillisecond)
  ->Args({100'000, 100'000})
  ->Args({100'000, 400'000})
  ->Args({100'000, 1'000'000})
  ->UseManualTime();

BENCHMARK_REGISTER_F(ConditionalJoin, conditional_inner_join_32bit_nulls)
  ->Unit(benchmark::kMillisecond)
  ->Args({100'000, 100'000})
  ->Args({100'000, 400'000})
  ->Args({100'000, 1'000'000})
  ->UseManualTime();

BENCHMARK_REGISTER_F(ConditionalJoin, conditional_inner_join_64bit_nulls)
  ->Unit(benchmark::kMillisecond)
  ->Args({100'000, 100'000})
  ->Args({100'000, 400'000})
  ->Args({100'000, 1'000'000})
  ->UseManualTime();

// left join -----------------------------------------------------------------------
BENCHMARK_REGISTER_F(ConditionalJoin, conditional_left_join_32bit)
  ->Unit(benchmark::kMillisecond)
  ->Args({100'000, 100'000})
  ->Args({100'000, 400'000})
  ->Args({100'000, 1'000'000})
  ->UseManualTime();

BENCHMARK_REGISTER_F(ConditionalJoin, conditional_left_join_64bit)
  ->Unit(benchmark::kMillisecond)
  ->Args({100'000, 100'000})
  ->Args({100'000, 400'000})
  ->Args({100'000, 1'000'000})
  ->UseManualTime();

BENCHMARK_REGISTER_F(ConditionalJoin, conditional_left_join_32bit_nulls)
  ->Unit(benchmark::kMillisecond)
  ->Args({100'000, 100'000})
  ->Args({100'000, 400'000})
  ->Args({100'000, 1'000'000})
  ->UseManualTime();

BENCHMARK_REGISTER_F(ConditionalJoin, conditional_left_join_64bit_nulls)
  ->Unit(benchmark::kMillisecond)
  ->Args({100'000, 100'000})
  ->Args({100'000, 400'000})
  ->Args({100'000, 1'000'000})
  ->UseManualTime();

// full join -----------------------------------------------------------------------
BENCHMARK_REGISTER_F(ConditionalJoin, conditional_full_join_32bit)
  ->Unit(benchmark::kMillisecond)
  ->Args({100'000, 100'000})
  ->Args({100'000, 400'000})
  ->Args({100'000, 1'000'000})
  ->UseManualTime();

BENCHMARK_REGISTER_F(ConditionalJoin, conditional_full_join_64bit)
  ->Unit(benchmark::kMillisecond)
  ->Args({100'000, 100'000})
  ->Args({100'000, 400'000})
  ->Args({100'000, 1'000'000})
  ->UseManualTime();

BENCHMARK_REGISTER_F(ConditionalJoin, conditional_full_join_32bit_nulls)
  ->Unit(benchmark::kMillisecond)
  ->Args({100'000, 100'000})
  ->Args({100'000, 400'000})
  ->Args({100'000, 1'000'000})
  ->UseManualTime();

BENCHMARK_REGISTER_F(ConditionalJoin, conditional_full_join_64bit_nulls)
  ->Unit(benchmark::kMillisecond)
  ->Args({100'000, 100'000})
  ->Args({100'000, 400'000})
  ->Args({100'000, 1'000'000})
  ->UseManualTime();

// left anti-join -------------------------------------------------------------
BENCHMARK_REGISTER_F(ConditionalJoin, conditional_left_anti_join_32bit)
  ->Unit(benchmark::kMillisecond)
  ->Args({100'000, 100'000})
  ->Args({100'000, 400'000})
  ->Args({100'000, 1'000'000})
  ->UseManualTime();

BENCHMARK_REGISTER_F(ConditionalJoin, conditional_left_anti_join_64bit)
  ->Unit(benchmark::kMillisecond)
  ->Args({100'000, 100'000})
  ->Args({100'000, 400'000})
  ->Args({100'000, 1'000'000})
  ->UseManualTime();

BENCHMARK_REGISTER_F(ConditionalJoin, conditional_left_anti_join_32bit_nulls)
  ->Unit(benchmark::kMillisecond)
  ->Args({100'000, 100'000})
  ->Args({100'000, 400'000})
  ->Args({100'000, 1'000'000})
  ->UseManualTime();

BENCHMARK_REGISTER_F(ConditionalJoin, conditional_left_anti_join_64bit_nulls)
  ->Unit(benchmark::kMillisecond)
  ->Args({100'000, 100'000})
  ->Args({100'000, 400'000})
  ->Args({100'000, 1'000'000})
  ->UseManualTime();

// left semi-join -------------------------------------------------------------
BENCHMARK_REGISTER_F(ConditionalJoin, conditional_left_semi_join_32bit)
  ->Unit(benchmark::kMillisecond)
  ->Args({100'000, 100'000})
  ->Args({100'000, 400'000})
  ->Args({100'000, 1'000'000})
  ->UseManualTime();

BENCHMARK_REGISTER_F(ConditionalJoin, conditional_left_semi_join_64bit)
  ->Unit(benchmark::kMillisecond)
  ->Args({100'000, 100'000})
  ->Args({100'000, 400'000})
  ->Args({100'000, 1'000'000})
  ->UseManualTime();

BENCHMARK_REGISTER_F(ConditionalJoin, conditional_left_semi_join_32bit_nulls)
  ->Unit(benchmark::kMillisecond)
  ->Args({100'000, 100'000})
  ->Args({100'000, 400'000})
  ->Args({100'000, 1'000'000})
  ->UseManualTime();

BENCHMARK_REGISTER_F(ConditionalJoin, conditional_left_semi_join_64bit_nulls)
  ->Unit(benchmark::kMillisecond)
  ->Args({100'000, 100'000})
  ->Args({100'000, 400'000})
  ->Args({100'000, 1'000'000})
  ->UseManualTime();
