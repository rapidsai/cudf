/*
 * Copyright (c) 2019, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <benchmark/benchmark.h>

#include <cudf/copying.hpp>

#include <cudf_test/base_fixture.hpp>
#include <cudf_test/cudf_gtest.hpp>
// #include <cudf_test/type_lists.hpp>
#include <cudf_test/column_utilities.hpp>
#include <cudf_test/column_wrapper.hpp>
#include <cudf_test/table_utilities.hpp>

#include <cudf/types.hpp>

#include <algorithm>
#include <random>

#include "../fixture/benchmark_fixture.hpp"
#include "../synchronization/synchronization.hpp"

class Scatter : public cudf::benchmark {
};

template <class TypeParam, bool coalesce>
void BM_scatter(benchmark::State& state)
{
  const cudf::size_type source_size{(cudf::size_type)state.range(0)};
  const cudf::size_type n_cols = (cudf::size_type)state.range(1);

  // Every element is valid
  auto data = cudf::detail::make_counting_transform_iterator(0, [](auto i) { return i; });

  // Gather indices
  std::vector<cudf::size_type> host_map_data(source_size);
  std::iota(host_map_data.begin(), host_map_data.end(), 0);

  if (coalesce) {
    std::reverse(host_map_data.begin(), host_map_data.end());
  } else {
    std::random_shuffle(host_map_data.begin(), host_map_data.end());
  }

  cudf::test::fixed_width_column_wrapper<cudf::size_type> scatter_map(host_map_data.begin(),
                                                                      host_map_data.end());

  std::vector<cudf::test::fixed_width_column_wrapper<TypeParam>> source_column_wrappers;
  std::vector<cudf::column_view> source_columns(n_cols);

  std::vector<cudf::test::fixed_width_column_wrapper<TypeParam>> target_column_wrappers;
  std::vector<cudf::column_view> target_columns(n_cols);

  std::generate_n(std::back_inserter(source_column_wrappers), n_cols, [=]() {
    return cudf::test::fixed_width_column_wrapper<TypeParam>(data, data + source_size);
  });
  std::transform(source_column_wrappers.begin(),
                 source_column_wrappers.end(),
                 source_columns.begin(),
                 [](auto const& col) { return static_cast<cudf::column_view>(col); });

  std::generate_n(std::back_inserter(target_column_wrappers), n_cols, [=]() {
    return cudf::test::fixed_width_column_wrapper<TypeParam>(data, data + source_size);
  });
  std::transform(target_column_wrappers.begin(),
                 target_column_wrappers.end(),
                 target_columns.begin(),
                 [](auto const& col) { return static_cast<cudf::column_view>(col); });

  cudf::table_view source_table{source_columns};
  cudf::table_view target_table{target_columns};

  for (auto _ : state) {
    cuda_event_timer raii(state, true);  // flush_l2_cache = true, stream = 0
    cudf::scatter(source_table, scatter_map, target_table);
  }

  state.SetBytesProcessed(static_cast<int64_t>(state.iterations()) * state.range(0) * n_cols * 2 *
                          sizeof(TypeParam));
}

#define SBM_BENCHMARK_DEFINE(name, type, coalesce)              \
  BENCHMARK_DEFINE_F(Scatter, name)(::benchmark::State & state) \
  {                                                             \
    BM_scatter<type, coalesce>(state);                          \
  }                                                             \
  BENCHMARK_REGISTER_F(Scatter, name)                           \
    ->RangeMultiplier(2)                                        \
    ->Ranges({{1 << 10, 1 << 25}, {1, 8}})                      \
    ->UseManualTime();

SBM_BENCHMARK_DEFINE(double_coalesce_x, double, true);
SBM_BENCHMARK_DEFINE(double_coalesce_o, double, false);
