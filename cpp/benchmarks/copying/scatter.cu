/*
 * Copyright (c) 2019-2022, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <benchmarks/common/generate_input.hpp>
#include <benchmarks/fixture/benchmark_fixture.hpp>
#include <benchmarks/synchronization/synchronization.hpp>

#include <cudf/copying.hpp>
#include <cudf/types.hpp>

#include <thrust/execution_policy.h>
#include <thrust/random.h>
#include <thrust/reverse.h>
#include <thrust/shuffle.h>

class Scatter : public cudf::benchmark {
};

template <class TypeParam, bool coalesce>
void BM_scatter(benchmark::State& state)
{
  auto const source_size{static_cast<cudf::size_type>(state.range(0))};
  auto const n_cols{static_cast<cudf::size_type>(state.range(1))};

  // Gather indices
  auto scatter_map_table =
    create_sequence_table({cudf::type_to_id<cudf::size_type>()}, row_count{source_size});
  auto scatter_map = scatter_map_table->get_column(0).mutable_view();

  if (coalesce) {
    thrust::reverse(
      thrust::device, scatter_map.begin<cudf::size_type>(), scatter_map.end<cudf::size_type>());
  } else {
    thrust::shuffle(thrust::device,
                    scatter_map.begin<cudf::size_type>(),
                    scatter_map.end<cudf::size_type>(),
                    thrust::default_random_engine());
  }

  // Every element is valid
  auto source_table = create_sequence_table(cycle_dtypes({cudf::type_to_id<TypeParam>()}, n_cols),
                                            row_count{source_size});
  auto target_table = create_sequence_table(cycle_dtypes({cudf::type_to_id<TypeParam>()}, n_cols),
                                            row_count{source_size});

  for (auto _ : state) {
    cuda_event_timer raii(state, true);  // flush_l2_cache = true, stream = 0
    cudf::scatter(*source_table, scatter_map, *target_table);
  }

  state.SetBytesProcessed(static_cast<int64_t>(state.iterations()) * state.range(0) * n_cols * 2 *
                          sizeof(TypeParam));
}

#define SBM_BENCHMARK_DEFINE(name, type, coalesce)              \
  BENCHMARK_DEFINE_F(Scatter, name)(::benchmark::State & state) \
  {                                                             \
    BM_scatter<type, coalesce>(state);                          \
  }                                                             \
  BENCHMARK_REGISTER_F(Scatter, name)                           \
    ->RangeMultiplier(2)                                        \
    ->Ranges({{1 << 10, 1 << 25}, {1, 8}})                      \
    ->UseManualTime();

SBM_BENCHMARK_DEFINE(double_coalesce_x, double, true);
SBM_BENCHMARK_DEFINE(double_coalesce_o, double, false);
