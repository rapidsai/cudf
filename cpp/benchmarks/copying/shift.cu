/*
 * Copyright (c) 2020-2023, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */
#include <benchmarks/common/generate_input.hpp>
#include <benchmarks/fixture/benchmark_fixture.hpp>
#include <benchmarks/synchronization/synchronization.hpp>

#include <cudf/copying.hpp>
#include <cudf/types.hpp>
#include <cudf/utilities/default_stream.hpp>

template <typename T, typename ScalarType = cudf::scalar_type_t<T>>
std::unique_ptr<cudf::scalar> make_scalar(
  T value                             = 0,
  rmm::cuda_stream_view stream        = cudf::get_default_stream(),
  rmm::mr::device_memory_resource* mr = rmm::mr::get_current_device_resource())
{
  auto s = new ScalarType(value, true, stream, mr);
  return std::unique_ptr<cudf::scalar>(s);
}

template <typename T>
struct value_func {
  T* data;
  cudf::size_type offset;

  __device__ T operator()(int idx) { return data[idx - offset]; }
};

struct validity_func {
  cudf::size_type size;
  cudf::size_type offset;

  __device__ bool operator()(int idx)
  {
    auto source_idx = idx - offset;
    return source_idx < 0 || source_idx >= size;
  }
};

template <bool use_validity, int shift_factor>
static void BM_shift(benchmark::State& state)
{
  cudf::size_type size   = state.range(0);
  cudf::size_type offset = size * (static_cast<double>(shift_factor) / 100.0);

  auto constexpr column_type_id = cudf::type_id::INT32;
  using column_type             = cudf::id_to_type<column_type_id>;

  auto const input_table = create_sequence_table(
    {column_type_id}, row_count{size}, use_validity ? std::optional<double>{1.0} : std::nullopt);
  cudf::column_view input{input_table->get_column(0)};

  auto fill = use_validity ? make_scalar<column_type>() : make_scalar<column_type>(777);

  for (auto _ : state) {
    cuda_event_timer raii(state, true);
    auto output = cudf::shift(input, offset, *fill);
  }

  auto const elems_read = (size - offset);
  auto const bytes_read = elems_read * sizeof(column_type);

  // If 'use_validity' is false, the fill value is a number, and the entire column
  // (excluding the null bitmask) needs to be written. On the other hand, if 'use_validity'
  // is true, only the elements that can be shifted are written, along with the full null bitmask.
  auto const elems_written = use_validity ? (size - offset) : size;
  auto const bytes_written = elems_written * sizeof(column_type);
  auto const null_bytes    = use_validity ? 2 * cudf::bitmask_allocation_size_bytes(size) : 0;

  state.SetBytesProcessed(static_cast<int64_t>(state.iterations()) *
                          (bytes_written + bytes_read + null_bytes));
}

class Shift : public cudf::benchmark {};

#define SHIFT_BM_BENCHMARK_DEFINE(name, use_validity, shift_factor) \
  BENCHMARK_DEFINE_F(Shift, name)(::benchmark::State & state)       \
  {                                                                 \
    BM_shift<use_validity, shift_factor>(state);                    \
  }                                                                 \
  BENCHMARK_REGISTER_F(Shift, name)                                 \
    ->RangeMultiplier(32)                                           \
    ->Range(1 << 10, 1 << 30)                                       \
    ->UseManualTime()                                               \
    ->Unit(benchmark::kMillisecond);

SHIFT_BM_BENCHMARK_DEFINE(shift_zero, false, 0);
SHIFT_BM_BENCHMARK_DEFINE(shift_zero_nullable_out, true, 0);

SHIFT_BM_BENCHMARK_DEFINE(shift_ten_percent, false, 10);
SHIFT_BM_BENCHMARK_DEFINE(shift_ten_percent_nullable_out, true, 10);

SHIFT_BM_BENCHMARK_DEFINE(shift_half, false, 50);
SHIFT_BM_BENCHMARK_DEFINE(shift_half_nullable_out, true, 50);

SHIFT_BM_BENCHMARK_DEFINE(shift_full, false, 100);
SHIFT_BM_BENCHMARK_DEFINE(shift_full_nullable_out, true, 100);
