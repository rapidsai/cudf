#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019-2022, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <benchmark/benchmark.h>

#include <cudf/column/column.hpp>

#include <cudf/copying.hpp>

#include <benchmarks/fixture/benchmark_fixture.hpp>
#include <benchmarks/synchronization/synchronization.hpp>
#include <cudf_test/column_wrapper.hpp>

// to enable, run cmake with -DBUILD_BENCHMARKS=ON

template <typename T>
void BM_contiguous_split_common(benchmark::State& state,
                                std::vector<T>& src_cols,
                                int64_t num_rows,
                                int64_t num_splits,
                                int64_t bytes_total)
{
  // generate splits
  std::vector<cudf::size_type> splits;
  if (num_splits > 0) {
    cudf::size_type const split_stride = num_rows / num_splits;
    // start after the first element.
    auto iter = thrust::make_counting_iterator(1);
    splits.reserve(num_splits);
    std::transform(iter,
                   iter + num_splits,
                   std::back_inserter(splits),
                   [split_stride, num_rows](cudf::size_type i) {
                     return std::min(i * split_stride, static_cast<cudf::size_type>(num_rows));
                   });
  }

  std::vector<std::unique_ptr<cudf::column>> columns(src_cols.size());
  std::transform(src_cols.begin(), src_cols.end(), columns.begin(), [](T& in) {
    auto ret = in.release();
    // computing the null count is not a part of the benchmark's target code path, and we want the
    // property to be pre-computed so that we measure the performance of only the intended code path
    [[maybe_unused]] auto const nulls = ret->null_count();
    return ret;
  });
  auto const src_table = cudf::table(std::move(columns));

  for (auto _ : state) {
    cuda_event_timer raii(state, true);  // flush_l2_cache = true, stream = 0
    auto result = cudf::contiguous_split(src_table, splits);
  }

  // it's 2x bytes_total because we're both reading and writing.
  state.SetBytesProcessed(static_cast<int64_t>(state.iterations()) * bytes_total * 2);
}

class ContiguousSplit : public cudf::benchmark {
};

void BM_contiguous_split(benchmark::State& state)
{
  int64_t const total_desired_bytes = state.range(0);
  cudf::size_type const num_cols    = state.range(1);
  cudf::size_type const num_splits  = state.range(2);
  bool const include_validity       = state.range(3) == 0 ? false : true;

  cudf::size_type el_size = 4;  // ints and floats
  int64_t const num_rows  = total_desired_bytes / (num_cols * el_size);

  // generate input table
  srand(31337);
  auto valids = cudf::detail::make_counting_transform_iterator(0, [](auto i) { return true; });
  std::vector<cudf::test::fixed_width_column_wrapper<int>> src_cols(num_cols);
  for (int idx = 0; idx < num_cols; idx++) {
    auto rand_elements =
      cudf::detail::make_counting_transform_iterator(0, [](int i) { return rand(); });
    if (include_validity) {
      src_cols[idx] = cudf::test::fixed_width_column_wrapper<int>(
        rand_elements, rand_elements + num_rows, valids);
    } else {
      src_cols[idx] =
        cudf::test::fixed_width_column_wrapper<int>(rand_elements, rand_elements + num_rows);
    }
  }

  int64_t const total_bytes =
    total_desired_bytes +
    (include_validity ? (max(int64_t{1}, (num_rows / 32)) * sizeof(cudf::bitmask_type) * num_cols)
                      : 0);

  BM_contiguous_split_common(state, src_cols, num_rows, num_splits, total_bytes);
}

class ContiguousSplitStrings : public cudf::benchmark {
};

int rand_range(int r)
{
  return static_cast<int>((static_cast<float>(rand()) / static_cast<float>(RAND_MAX)) *
                          (float)(r - 1));
}

void BM_contiguous_split_strings(benchmark::State& state)
{
  int64_t const total_desired_bytes = state.range(0);
  cudf::size_type const num_cols    = state.range(1);
  cudf::size_type const num_splits  = state.range(2);
  bool const include_validity       = state.range(3) == 0 ? false : true;

  constexpr int64_t string_len = 8;
  std::vector<const char*> h_strings{
    "aaaaaaaa", "bbbbbbbb", "cccccccc", "dddddddd", "eeeeeeee", "ffffffff", "gggggggg", "hhhhhhhh"};

  int64_t const col_len_bytes = total_desired_bytes / num_cols;
  int64_t const num_rows      = col_len_bytes / string_len;

  // generate input table
  srand(31337);
  auto valids = cudf::detail::make_counting_transform_iterator(
    0, [](auto i) { return i % 2 == 0 ? true : false; });
  std::vector<cudf::test::strings_column_wrapper> src_cols;
  std::vector<const char*> one_col(num_rows);
  for (int64_t idx = 0; idx < num_cols; idx++) {
    // fill in a random set of strings
    for (int64_t s_idx = 0; s_idx < num_rows; s_idx++) {
      one_col[s_idx] = h_strings[rand_range(h_strings.size())];
    }
    if (include_validity) {
      src_cols.push_back(
        cudf::test::strings_column_wrapper(one_col.begin(), one_col.end(), valids));
    } else {
      src_cols.push_back(cudf::test::strings_column_wrapper(one_col.begin(), one_col.end()));
    }
  }

  int64_t const total_bytes =
    total_desired_bytes + ((num_rows + 1) * sizeof(cudf::offset_type)) +
    (include_validity ? (max(int64_t{1}, (num_rows / 32)) * sizeof(cudf::bitmask_type) * num_cols)
                      : 0);

  BM_contiguous_split_common(state, src_cols, num_rows, num_splits, total_bytes);
}

#define CSBM_BENCHMARK_DEFINE(name, size, num_columns, num_splits, validity) \
  BENCHMARK_DEFINE_F(ContiguousSplit, name)(::benchmark::State & state)      \
  {                                                                          \
    BM_contiguous_split(state);                                              \
  }                                                                          \
  BENCHMARK_REGISTER_F(ContiguousSplit, name)                                \
    ->Args({size, num_columns, num_splits, validity})                        \
    ->Unit(benchmark::kMillisecond)                                          \
    ->UseManualTime()                                                        \
    ->Iterations(8)
CSBM_BENCHMARK_DEFINE(6Gb512ColsNoValidity, (int64_t)6 * 1024 * 1024 * 1024, 512, 256, 0);
CSBM_BENCHMARK_DEFINE(6Gb512ColsValidity, (int64_t)6 * 1024 * 1024 * 1024, 512, 256, 1);
CSBM_BENCHMARK_DEFINE(6Gb10ColsNoValidity, (int64_t)6 * 1024 * 1024 * 1024, 10, 256, 0);
CSBM_BENCHMARK_DEFINE(6Gb10ColsValidity, (int64_t)6 * 1024 * 1024 * 1024, 10, 256, 1);

CSBM_BENCHMARK_DEFINE(4Gb512ColsNoValidity, (int64_t)4 * 1024 * 1024 * 1024, 512, 256, 0);
CSBM_BENCHMARK_DEFINE(4Gb512ColsValidity, (int64_t)4 * 1024 * 1024 * 1024, 512, 256, 1);
CSBM_BENCHMARK_DEFINE(4Gb10ColsNoValidity, (int64_t)4 * 1024 * 1024 * 1024, 10, 256, 0);
CSBM_BENCHMARK_DEFINE(4Gb10ColsValidity, (int64_t)4 * 1024 * 1024 * 1024, 10, 256, 1);
CSBM_BENCHMARK_DEFINE(4Gb4ColsNoSplits, (int64_t)1 * 1024 * 1024 * 1024, 4, 0, 1);
CSBM_BENCHMARK_DEFINE(4Gb4ColsValidityNoSplits, (int64_t)1 * 1024 * 1024 * 1024, 4, 0, 1);

CSBM_BENCHMARK_DEFINE(1Gb512ColsNoValidity, (int64_t)1 * 1024 * 1024 * 1024, 512, 256, 0);
CSBM_BENCHMARK_DEFINE(1Gb512ColsValidity, (int64_t)1 * 1024 * 1024 * 1024, 512, 256, 1);
CSBM_BENCHMARK_DEFINE(1Gb10ColsNoValidity, (int64_t)1 * 1024 * 1024 * 1024, 10, 256, 0);
CSBM_BENCHMARK_DEFINE(1Gb10ColsValidity, (int64_t)1 * 1024 * 1024 * 1024, 10, 256, 1);
CSBM_BENCHMARK_DEFINE(1Gb1ColNoSplits, (int64_t)1 * 1024 * 1024 * 1024, 1, 0, 1);
CSBM_BENCHMARK_DEFINE(1Gb1ColValidityNoSplits, (int64_t)1 * 1024 * 1024 * 1024, 1, 0, 1);

#define CSBM_STRINGS_BENCHMARK_DEFINE(name, size, num_columns, num_splits, validity) \
  BENCHMARK_DEFINE_F(ContiguousSplitStrings, name)(::benchmark::State & state)       \
  {                                                                                  \
    BM_contiguous_split_strings(state);                                              \
  }                                                                                  \
  BENCHMARK_REGISTER_F(ContiguousSplitStrings, name)                                 \
    ->Args({size, num_columns, num_splits, validity})                                \
    ->Unit(benchmark::kMillisecond)                                                  \
    ->UseManualTime()                                                                \
    ->Iterations(8)

CSBM_STRINGS_BENCHMARK_DEFINE(4Gb512ColsNoValidity, (int64_t)4 * 1024 * 1024 * 1024, 512, 256, 0);
CSBM_STRINGS_BENCHMARK_DEFINE(4Gb512ColsValidity, (int64_t)4 * 1024 * 1024 * 1024, 512, 256, 1);
CSBM_STRINGS_BENCHMARK_DEFINE(4Gb10ColsNoValidity, (int64_t)4 * 1024 * 1024 * 1024, 10, 256, 0);
CSBM_STRINGS_BENCHMARK_DEFINE(4Gb10ColsValidity, (int64_t)4 * 1024 * 1024 * 1024, 10, 256, 1);
CSBM_STRINGS_BENCHMARK_DEFINE(4Gb4ColsNoSplits, (int64_t)1 * 1024 * 1024 * 1024, 4, 0, 0);
CSBM_STRINGS_BENCHMARK_DEFINE(4Gb4ColsValidityNoSplits, (int64_t)1 * 1024 * 1024 * 1024, 4, 0, 1);

CSBM_STRINGS_BENCHMARK_DEFINE(1Gb512ColsNoValidity, (int64_t)1 * 1024 * 1024 * 1024, 512, 256, 0);
CSBM_STRINGS_BENCHMARK_DEFINE(1Gb512ColsValidity, (int64_t)1 * 1024 * 1024 * 1024, 512, 256, 1);
CSBM_STRINGS_BENCHMARK_DEFINE(1Gb10ColsNoValidity, (int64_t)1 * 1024 * 1024 * 1024, 10, 256, 0);
CSBM_STRINGS_BENCHMARK_DEFINE(1Gb10ColsValidity, (int64_t)1 * 1024 * 1024 * 1024, 10, 256, 1);
CSBM_STRINGS_BENCHMARK_DEFINE(1Gb1ColNoSplits, (int64_t)1 * 1024 * 1024 * 1024, 1, 0, 0);
CSBM_STRINGS_BENCHMARK_DEFINE(1Gb1ColValidityNoSplits, (int64_t)1 * 1024 * 1024 * 1024, 1, 0, 1);
