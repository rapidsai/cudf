#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <benchmark/benchmark.h>

#include <cudf/column/column.hpp>

#include <cudf/copying.hpp>

#include <benchmarks/fixture/benchmark_fixture.hpp>
#include <benchmarks/synchronization/synchronization.hpp>
#include <cudf_test/column_wrapper.hpp>

// to enable, run cmake with -DBUILD_BENCHMARKS=ON

template <typename T>
void BM_contiguous_split_common(benchmark::State& state,
                                std::vector<T>& src_cols,
                                int64_t num_rows,
                                int64_t num_splits,
                                int64_t bytes_total)
{
  // generate splits
  cudf::size_type split_stride = num_rows / num_splits;
  std::vector<cudf::size_type> splits;
  for (int idx = 0; idx < num_rows; idx += split_stride) {
    splits.push_back(std::min(idx + split_stride, static_cast<cudf::size_type>(num_rows)));
  }

  std::vector<std::unique_ptr<cudf::column>> columns(src_cols.size());
  std::transform(src_cols.begin(), src_cols.end(), columns.begin(), [](T& in) {
    auto ret = in.release();
    ret->null_count();
    return ret;
  });
  cudf::table src_table(std::move(columns));

  for (auto _ : state) {
    cuda_event_timer raii(state, true);  // flush_l2_cache = true, stream = 0
    auto result = cudf::contiguous_split(src_table, splits);
  }

  state.SetBytesProcessed(static_cast<int64_t>(state.iterations()) * bytes_total);
}

class ContiguousSplit : public cudf::benchmark {
};

void BM_contiguous_split(benchmark::State& state)
{
  int64_t total_desired_bytes = state.range(0);
  cudf::size_type num_cols    = state.range(1);
  cudf::size_type num_splits  = state.range(2);
  bool include_validity       = state.range(3) == 0 ? false : true;

  cudf::size_type el_size = 4;  // ints and floats
  int64_t num_rows        = total_desired_bytes / (num_cols * el_size);

  // generate input table
  srand(31337);
  auto valids = cudf::test::make_counting_transform_iterator(0, [](auto i) { return true; });
  std::vector<cudf::test::fixed_width_column_wrapper<int>> src_cols(num_cols);
  for (int idx = 0; idx < num_cols; idx++) {
    auto rand_elements =
      cudf::test::make_counting_transform_iterator(0, [](int i) { return rand(); });
    if (include_validity) {
      src_cols[idx] = cudf::test::fixed_width_column_wrapper<int>(
        rand_elements, rand_elements + num_rows, valids);
    } else {
      src_cols[idx] =
        cudf::test::fixed_width_column_wrapper<int>(rand_elements, rand_elements + num_rows);
    }
  }

  size_t total_bytes = total_desired_bytes;
  if (include_validity) { total_bytes += num_rows / (sizeof(cudf::bitmask_type) * 8); }

  BM_contiguous_split_common(state, src_cols, num_rows, num_splits, total_bytes);
}

class ContiguousSplitStrings : public cudf::benchmark {
};

int rand_range(int r)
{
  return static_cast<int>((static_cast<float>(rand()) / static_cast<float>(RAND_MAX)) *
                          (float)(r - 1));
}

void BM_contiguous_split_strings(benchmark::State& state)
{
  int64_t total_desired_bytes = state.range(0);
  cudf::size_type num_cols    = state.range(1);
  cudf::size_type num_splits  = state.range(2);
  bool include_validity       = state.range(3) == 0 ? false : true;

  const int64_t string_len = 8;
  std::vector<const char*> h_strings{
    "aaaaaaaa", "bbbbbbbb", "cccccccc", "dddddddd", "eeeeeeee", "ffffffff", "gggggggg", "hhhhhhhh"};

  int64_t col_len_bytes = total_desired_bytes / num_cols;
  int64_t num_rows      = col_len_bytes / string_len;

  // generate input table
  srand(31337);
  auto valids = cudf::test::make_counting_transform_iterator(
    0, [](auto i) { return i % 2 == 0 ? true : false; });
  std::vector<cudf::test::strings_column_wrapper> src_cols;
  std::vector<const char*> one_col(num_rows);
  for (int64_t idx = 0; idx < num_cols; idx++) {
    // fill in a random set of strings
    for (int64_t s_idx = 0; s_idx < num_rows; s_idx++) {
      one_col[s_idx] = h_strings[rand_range(h_strings.size())];
    }
    if (include_validity) {
      src_cols.push_back(
        cudf::test::strings_column_wrapper(one_col.begin(), one_col.end(), valids));
    } else {
      src_cols.push_back(cudf::test::strings_column_wrapper(one_col.begin(), one_col.end()));
    }
  }

  size_t total_bytes = total_desired_bytes + (num_rows * sizeof(cudf::size_type));
  if (include_validity) { total_bytes += num_rows / (sizeof(cudf::bitmask_type) * 8); }

  BM_contiguous_split_common(state, src_cols, num_rows, num_splits, total_bytes);
}

#define CSBM_BENCHMARK_DEFINE(name, size, num_columns, num_splits, validity) \
  BENCHMARK_DEFINE_F(ContiguousSplit, name)(::benchmark::State & state)      \
  {                                                                          \
    BM_contiguous_split(state);                                              \
  }                                                                          \
  BENCHMARK_REGISTER_F(ContiguousSplit, name)                                \
    ->Args({size, num_columns, num_splits, validity})                        \
    ->Unit(benchmark::kMillisecond)                                          \
    ->UseManualTime()                                                        \
    ->Iterations(8)
CSBM_BENCHMARK_DEFINE(6Gb512ColsNoValidity, (int64_t)6 * 1024 * 1024 * 1024, 512, 256, 0);
CSBM_BENCHMARK_DEFINE(6Gb512ColsValidity, (int64_t)6 * 1024 * 1024 * 1024, 512, 256, 1);
CSBM_BENCHMARK_DEFINE(6Gb10ColsNoValidity, (int64_t)6 * 1024 * 1024 * 1024, 10, 256, 0);
CSBM_BENCHMARK_DEFINE(6Gb10ColsValidity, (int64_t)6 * 1024 * 1024 * 1024, 10, 256, 1);

CSBM_BENCHMARK_DEFINE(4Gb512ColsNoValidity, (int64_t)4 * 1024 * 1024 * 1024, 512, 256, 0);
CSBM_BENCHMARK_DEFINE(4Gb512ColsValidity, (int64_t)4 * 1024 * 1024 * 1024, 512, 256, 1);
CSBM_BENCHMARK_DEFINE(4Gb10ColsNoValidity, (int64_t)4 * 1024 * 1024 * 1024, 10, 256, 0);
CSBM_BENCHMARK_DEFINE(46b10ColsValidity, (int64_t)4 * 1024 * 1024 * 1024, 10, 256, 1);

CSBM_BENCHMARK_DEFINE(1Gb512ColsNoValidity, (int64_t)1 * 1024 * 1024 * 1024, 512, 256, 0);
CSBM_BENCHMARK_DEFINE(1Gb512ColsValidity, (int64_t)1 * 1024 * 1024 * 1024, 512, 256, 1);
CSBM_BENCHMARK_DEFINE(1Gb10ColsNoValidity, (int64_t)1 * 1024 * 1024 * 1024, 10, 256, 0);
CSBM_BENCHMARK_DEFINE(1Gb10ColsValidity, (int64_t)1 * 1024 * 1024 * 1024, 10, 256, 1);

#define CSBM_STRINGS_BENCHMARK_DEFINE(name, size, num_columns, num_splits, validity) \
  BENCHMARK_DEFINE_F(ContiguousSplitStrings, name)(::benchmark::State & state)       \
  {                                                                                  \
    BM_contiguous_split_strings(state);                                              \
  }                                                                                  \
  BENCHMARK_REGISTER_F(ContiguousSplitStrings, name)                                 \
    ->Args({size, num_columns, num_splits, validity})                                \
    ->Unit(benchmark::kMillisecond)                                                  \
    ->UseManualTime()                                                                \
    ->Iterations(8)

CSBM_STRINGS_BENCHMARK_DEFINE(4Gb512ColsNoValidity, (int64_t)4 * 1024 * 1024 * 1024, 512, 256, 0);
CSBM_STRINGS_BENCHMARK_DEFINE(4Gb512ColsValidity, (int64_t)4 * 1024 * 1024 * 1024, 512, 256, 1);
CSBM_STRINGS_BENCHMARK_DEFINE(4Gb10ColsNoValidity, (int64_t)4 * 1024 * 1024 * 1024, 10, 256, 0);
CSBM_STRINGS_BENCHMARK_DEFINE(4Gb10ColsValidity, (int64_t)4 * 1024 * 1024 * 1024, 10, 256, 1);

CSBM_STRINGS_BENCHMARK_DEFINE(1Gb512ColsNoValidity, (int64_t)1 * 1024 * 1024 * 1024, 512, 256, 0);
CSBM_STRINGS_BENCHMARK_DEFINE(1Gb512ColsValidity, (int64_t)1 * 1024 * 1024 * 1024, 512, 256, 1);
CSBM_STRINGS_BENCHMARK_DEFINE(1Gb10ColsNoValidity, (int64_t)1 * 1024 * 1024 * 1024, 10, 256, 0);
CSBM_STRINGS_BENCHMARK_DEFINE(1Gb10ColsValidity, (int64_t)1 * 1024 * 1024 * 1024, 10, 256, 1);
