#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <benchmark/benchmark.h>

#include <cudf_test/column_wrapper.hpp>
#include <random>

#include "../fixture/benchmark_fixture.hpp"
#include "../synchronization/synchronization.hpp"

#include <cudf/detail/iterator.cuh>  // include iterator header
// for reduction tests
#include <thrust/device_vector.h>
#include <cub/device/device_reduce.cuh>
#include <cudf/detail/utilities/device_operators.cuh>

template <typename T>
T random_int(T min, T max)
{
  static unsigned seed = 13377331;
  static std::mt19937 engine{seed};
  static std::uniform_int_distribution<T> uniform{min, max};

  return uniform(engine);
}

// -----------------------------------------------------------------------------
template <typename InputIterator, typename OutputIterator, typename T>
inline auto reduce_by_cub(OutputIterator result, InputIterator d_in, int num_items, T init)
{
  size_t temp_storage_bytes = 0;

  hipcub::DeviceReduce::Reduce(
    nullptr, temp_storage_bytes, d_in, result, num_items, cudf::DeviceSum{}, init);

  // Allocate temporary storage
  rmm::device_buffer d_temp_storage(temp_storage_bytes);

  // Run reduction
  hipcub::DeviceReduce::Reduce(
    d_temp_storage.data(), temp_storage_bytes, d_in, result, num_items, cudf::DeviceSum{}, init);

  return temp_storage_bytes;
}

// -----------------------------------------------------------------------------
template <typename T>
void raw_stream_bench_cub(cudf::column_view &col, rmm::device_vector<T> &result)
{
  // std::cout << "raw stream cub: " << "\t";

  T init{0};
  auto begin    = col.data<T>();
  int num_items = col.size();

  reduce_by_cub(result.begin(), begin, num_items, init);
};

template <typename T, bool has_null>
void iterator_bench_cub(cudf::column_view &col, rmm::device_vector<T> &result)
{
  // std::cout << "iterator cub " << ( (has_null) ? "<true>: " : "<false>: " ) << "\t";

  T init{0};
  auto d_col    = cudf::column_device_view::create(col);
  int num_items = col.size();
  if (has_null) {
    auto begin = cudf::detail::make_null_replacement_iterator(*d_col, init);
    reduce_by_cub(result.begin(), begin, num_items, init);
  } else {
    auto begin = d_col->begin<T>();
    reduce_by_cub(result.begin(), begin, num_items, init);
  }
}

// -----------------------------------------------------------------------------
template <typename T>
void raw_stream_bench_thrust(cudf::column_view &col, rmm::device_vector<T> &result)
{
  // std::cout << "raw stream thust: " << "\t\t";

  T init{0};
  auto d_in  = col.data<T>();
  auto d_end = d_in + col.size();
  thrust::reduce(thrust::device, d_in, d_end, init, cudf::DeviceSum{});
}

template <typename T, bool has_null>
void iterator_bench_thrust(cudf::column_view &col, rmm::device_vector<T> &result)
{
  // std::cout << "iterator thust " << ( (has_null) ? "<true>: " : "<false>: " ) << "\t";

  T init{0};
  auto d_col = cudf::column_device_view::create(col);
  if (has_null) {
    auto d_in  = cudf::detail::make_null_replacement_iterator(*d_col, init);
    auto d_end = d_in + col.size();
    thrust::reduce(thrust::device, d_in, d_end, init, cudf::DeviceSum{});
  } else {
    auto d_in  = d_col->begin<T>();
    auto d_end = d_in + col.size();
    thrust::reduce(thrust::device, d_in, d_end, init, cudf::DeviceSum{});
  }
}

// -----------------------------------------------------------------------------
class Iterator : public cudf::benchmark {
};

template <class TypeParam, bool cub_or_thrust, bool raw_or_iterator>
void BM_iterator(benchmark::State &state)
{
  const cudf::size_type column_size{(cudf::size_type)state.range(0)};
  using T      = TypeParam;
  auto num_gen = thrust::counting_iterator<cudf::size_type>(0);

  cudf::test::fixed_width_column_wrapper<T> wrap_hasnull_F(num_gen, num_gen + column_size);
  cudf::column_view hasnull_F = wrap_hasnull_F;

  rmm::device_vector<T> dev_result(1, T{0});
  for (auto _ : state) {
    cuda_event_timer raii(state, true);  // flush_l2_cache = true, stream = 0
    if (cub_or_thrust) {
      if (raw_or_iterator) {
        raw_stream_bench_cub<T>(hasnull_F, dev_result);  // driven by raw pointer
      } else {
        iterator_bench_cub<T, false>(hasnull_F, dev_result);  // driven by riterator without nulls
      }
    } else {
      if (raw_or_iterator) {
        raw_stream_bench_thrust<T>(hasnull_F, dev_result);  // driven by raw pointer
      } else {
        iterator_bench_thrust<T, false>(hasnull_F,
                                        dev_result);  // driven by riterator without nulls
      }
    }
  }
  state.SetBytesProcessed(static_cast<int64_t>(state.iterations()) * column_size *
                          sizeof(TypeParam));
}

// operator+ defined for pair iterator reduction
template <typename T>
__device__ thrust::pair<T, bool> operator+(thrust::pair<T, bool> lhs, thrust::pair<T, bool> rhs)
{
  return thrust::pair<T, bool>{lhs.first * lhs.second + rhs.first * rhs.second,
                               lhs.second + rhs.second};
}
// -----------------------------------------------------------------------------
template <typename T, bool has_null>
void pair_iterator_bench_cub(cudf::column_view &col,
                             rmm::device_vector<thrust::pair<T, bool>> &result)
{
  thrust::pair<T, bool> init{0, false};
  auto d_col    = cudf::column_device_view::create(col);
  int num_items = col.size();
  auto begin    = d_col->pair_begin<T, has_null>();
  reduce_by_cub(result.begin(), begin, num_items, init);
}

template <typename T, bool has_null>
void pair_iterator_bench_thrust(cudf::column_view &col,
                                rmm::device_vector<thrust::pair<T, bool>> &result)
{
  thrust::pair<T, bool> init{0, false};
  auto d_col = cudf::column_device_view::create(col);
  auto d_in  = d_col->pair_begin<T, has_null>();
  auto d_end = d_in + col.size();
  thrust::reduce(thrust::device, d_in, d_end, init, cudf::DeviceSum{});
}

template <class TypeParam, bool cub_or_thrust>
void BM_pair_iterator(benchmark::State &state)
{
  const cudf::size_type column_size{(cudf::size_type)state.range(0)};
  using T      = TypeParam;
  auto num_gen = thrust::counting_iterator<cudf::size_type>(0);
  auto null_gen =
    thrust::make_transform_iterator(num_gen, [](cudf::size_type row) { return row % 2 == 0; });

  cudf::test::fixed_width_column_wrapper<T> wrap_hasnull_F(num_gen, num_gen + column_size);
  cudf::test::fixed_width_column_wrapper<T> wrap_hasnull_T(
    num_gen, num_gen + column_size, null_gen);
  cudf::column_view hasnull_F = wrap_hasnull_F;
  cudf::column_view hasnull_T = wrap_hasnull_T;

  rmm::device_vector<thrust::pair<T, bool>> dev_result(1, {T{0}, false});
  for (auto _ : state) {
    cuda_event_timer raii(state, true);  // flush_l2_cache = true, stream = 0
    if (cub_or_thrust) {
      pair_iterator_bench_cub<T, false>(hasnull_T,
                                        dev_result);  // driven by pair iterator with nulls
    } else {
      pair_iterator_bench_thrust<T, false>(hasnull_T,
                                           dev_result);  // driven by pair iterator with nulls
    }
  }
  state.SetBytesProcessed(static_cast<int64_t>(state.iterations()) * column_size *
                          sizeof(TypeParam));
}

#define ITER_BM_BENCHMARK_DEFINE(name, type, cub_or_thrust, raw_or_iterator) \
  BENCHMARK_DEFINE_F(Iterator, name)(::benchmark::State & state)             \
  {                                                                          \
    BM_iterator<type, cub_or_thrust, raw_or_iterator>(state);                \
  }                                                                          \
  BENCHMARK_REGISTER_F(Iterator, name)                                       \
    ->RangeMultiplier(10)                                                    \
    ->Range(1000, 10000000)                                                  \
    ->UseManualTime()                                                        \
    ->Unit(benchmark::kMillisecond);

ITER_BM_BENCHMARK_DEFINE(double_cub_raw, double, true, true);
ITER_BM_BENCHMARK_DEFINE(double_cub_iter, double, true, false);
ITER_BM_BENCHMARK_DEFINE(double_thrust_raw, double, false, true);
ITER_BM_BENCHMARK_DEFINE(double_thrust_iter, double, false, false);

#define PAIRITER_BM_BENCHMARK_DEFINE(name, type, cub_or_thrust)  \
  BENCHMARK_DEFINE_F(Iterator, name)(::benchmark::State & state) \
  {                                                              \
    BM_pair_iterator<type, cub_or_thrust>(state);                \
  }                                                              \
  BENCHMARK_REGISTER_F(Iterator, name)                           \
    ->RangeMultiplier(10)                                        \
    ->Range(1000, 10000000)                                      \
    ->UseManualTime()                                            \
    ->Unit(benchmark::kMillisecond);

PAIRITER_BM_BENCHMARK_DEFINE(double_cub_pair, double, true);
PAIRITER_BM_BENCHMARK_DEFINE(double_thrust_pair, double, false);
