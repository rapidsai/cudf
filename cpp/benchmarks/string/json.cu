#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2021-2022, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <benchmarks/common/generate_input.hpp>
#include <benchmarks/fixture/benchmark_fixture.hpp>
#include <benchmarks/synchronization/synchronization.hpp>

#include <cudf_test/column_wrapper.hpp>

#include <cudf/column/column_factories.hpp>
#include <cudf/strings/detail/strings_children.cuh>
#include <cudf/strings/detail/utilities.cuh>
#include <cudf/strings/json.hpp>
#include <cudf/strings/string_view.hpp>
#include <cudf/strings/strings_column_view.hpp>
#include <cudf/types.hpp>

#include <thrust/random.h>

class JsonPath : public cudf::benchmark {
};

const std::vector<std::string> Books{
  R"json({
"category": "reference",
"author": "Nigel Rees",
"title": "Sayings of the Century",
"price": 8.95
})json",
  R"json({
"category": "fiction",
"author": "Evelyn Waugh",
"title": "Sword of Honour",
"price": 12.99
})json",
  R"json({
"category": "fiction",
"author": "Herman Melville",
"title": "Moby Dick",
"isbn": "0-553-21311-3",
"price": 8.99
})json",
  R"json({
"category": "fiction",
"author": "J. R. R. Tolkien",
"title": "The Lord of the Rings",
"isbn": "0-395-19395-8",
"price": 22.99
})json"};
constexpr int Approx_book_size = 110;
const std::vector<std::string> Bicycles{
  R"json({"color": "red", "price": 9.95})json",
  R"json({"color": "green", "price": 29.95})json",
  R"json({"color": "blue", "price": 399.95})json",
  R"json({"color": "yellow", "price": 99.95})json",
  R"json({"color": "mauve", "price": 199.95})json",
};
constexpr int Approx_bicycle_size = 33;
std::string Misc{"\n\"expensive\": 10\n"};

struct json_benchmark_row_builder {
  int const desired_bytes;
  cudf::size_type const num_rows;
  cudf::column_device_view const d_books_bicycles[2];  // Books, Bicycles strings
  cudf::column_device_view const d_book_pct;           // Book percentage
  cudf::column_device_view const d_misc_order;         // Misc-Store order
  cudf::column_device_view const d_store_order;        // Books-Bicycles order
  int32_t* d_offsets{};
  char* d_chars{};
  thrust::minstd_rand rng{5236};
  thrust::uniform_int_distribution<int> dist{};

  // internal data structure for {bytes, out_ptr} with operator+=
  struct bytes_and_ptr {
    cudf::size_type bytes;
    char* ptr;
    __device__ bytes_and_ptr& operator+=(cudf::string_view const& str_append)
    {
      bytes += str_append.size_bytes();
      if (ptr) { ptr = cudf::strings::detail::copy_string(ptr, str_append); }
      return *this;
    }
  };

  __device__ inline void copy_items(int this_idx,
                                    cudf::size_type num_items,
                                    bytes_and_ptr& output_str)
  {
    using param_type = thrust::uniform_int_distribution<int>::param_type;
    dist.param(param_type{0, d_books_bicycles[this_idx].size() - 1});
    cudf::string_view comma(",\n", 2);
    for (int i = 0; i < num_items; i++) {
      if (i > 0) { output_str += comma; }
      int idx   = dist(rng);
      auto item = d_books_bicycles[this_idx].element<cudf::string_view>(idx);
      output_str += item;
    }
  }

  __device__ void operator()(cudf::size_type idx)
  {
    int num_books       = 2;
    int num_bicycles    = 2;
    int remaining_bytes = max(
      0, desired_bytes - ((num_books * Approx_book_size) + (num_bicycles * Approx_bicycle_size)));

    // divide up the remainder between books and bikes
    auto book_pct = d_book_pct.element<float>(idx);
    // {Misc, store} OR {store, Misc}
    // store: {books, bicycles} OR store: {bicycles, books}
    float bicycle_pct = 1.0f - book_pct;
    num_books += (remaining_bytes * book_pct) / Approx_book_size;
    num_bicycles += (remaining_bytes * bicycle_pct) / Approx_bicycle_size;

    char* out_ptr = d_chars ? d_chars + d_offsets[idx] : nullptr;
    bytes_and_ptr output_str{0, out_ptr};
    //
    cudf::string_view comma(",\n", 2);
    cudf::string_view brace1("{\n", 2);
    cudf::string_view store_member_start[2]{{"\"book\": [\n", 10}, {"\"bicycle\": [\n", 13}};
    cudf::string_view store("\"store\": {\n", 11);
    cudf::string_view Misc{"\"expensive\": 10", 15};
    cudf::string_view brace2("\n}", 2);
    cudf::string_view square2{"\n]", 2};

    output_str += brace1;
    if (d_misc_order.element<bool>(idx)) {  // Misc. first.
      output_str += Misc;
      output_str += comma;
    }
    output_str += store;
    for (int store_order = 0; store_order < 2; store_order++) {
      if (store_order > 0) { output_str += comma; }
      int this_idx    = (d_store_order.element<bool>(idx) == store_order);
      auto& mem_start = store_member_start[this_idx];
      output_str += mem_start;
      copy_items(this_idx, this_idx == 0 ? num_books : num_bicycles, output_str);
      output_str += square2;
    }
    output_str += brace2;
    if (!d_misc_order.element<bool>(idx)) {  // Misc, if not first.
      output_str += comma;
      output_str += Misc;
    }
    output_str += brace2;
    if (!output_str.ptr) d_offsets[idx] = output_str.bytes;
  }
};

auto build_json_string_column(int desired_bytes, int num_rows)
{
  data_profile const profile = data_profile_builder().cardinality(0).no_validity().distribution(
    cudf::type_id::FLOAT32, distribution_id::UNIFORM, 0.0, 1.0);
  auto float_2bool_columns =
    create_random_table({cudf::type_id::FLOAT32, cudf::type_id::BOOL8, cudf::type_id::BOOL8},
                        row_count{num_rows},
                        profile);

  cudf::test::strings_column_wrapper books(Books.begin(), Books.end());
  cudf::test::strings_column_wrapper bicycles(Bicycles.begin(), Bicycles.end());
  auto d_books       = cudf::column_device_view::create(books);
  auto d_bicycles    = cudf::column_device_view::create(bicycles);
  auto d_book_pct    = cudf::column_device_view::create(float_2bool_columns->get_column(0));
  auto d_misc_order  = cudf::column_device_view::create(float_2bool_columns->get_column(1));
  auto d_store_order = cudf::column_device_view::create(float_2bool_columns->get_column(2));
  json_benchmark_row_builder jb{
    desired_bytes, num_rows, {*d_books, *d_bicycles}, *d_book_pct, *d_misc_order, *d_store_order};
  auto children = cudf::strings::detail::make_strings_children(
    jb, num_rows, cudf::get_default_stream(), rmm::mr::get_current_device_resource());
  return cudf::make_strings_column(
    num_rows, std::move(children.first), std::move(children.second), 0, {});
}

void BM_case(benchmark::State& state, std::string query_arg)
{
  srand(5236);
  int num_rows      = state.range(0);
  int desired_bytes = state.range(1);
  auto input        = build_json_string_column(desired_bytes, num_rows);
  cudf::strings_column_view scv(input->view());
  size_t num_chars = scv.chars().size();

  std::string json_path(query_arg);

  for (auto _ : state) {
    cuda_event_timer raii(state, true);
    auto result = cudf::strings::get_json_object(scv, json_path);
    hipStreamSynchronize(0);
  }

  // this isn't strictly 100% accurate. a given query isn't necessarily
  // going to visit every single incoming character.  but in spirit it does.
  state.SetBytesProcessed(state.iterations() * num_chars);
}

#define JSON_BENCHMARK_DEFINE(name, query)                                                  \
  BENCHMARK_DEFINE_F(JsonPath, name)(::benchmark::State & state) { BM_case(state, query); } \
  BENCHMARK_REGISTER_F(JsonPath, name)                                                      \
    ->ArgsProduct({{100, 1000, 100000, 400000}, {300, 600, 4096}})                          \
    ->UseManualTime()                                                                       \
    ->Unit(benchmark::kMillisecond);

JSON_BENCHMARK_DEFINE(query0, "$");
JSON_BENCHMARK_DEFINE(query1, "$.store");
JSON_BENCHMARK_DEFINE(query2, "$.store.book");
JSON_BENCHMARK_DEFINE(query3, "$.store.*");
JSON_BENCHMARK_DEFINE(query4, "$.store.book[*]");
JSON_BENCHMARK_DEFINE(query5, "$.store.book[*].category");
JSON_BENCHMARK_DEFINE(query6, "$.store['bicycle']");
JSON_BENCHMARK_DEFINE(query7, "$.store.book[*]['isbn']");
JSON_BENCHMARK_DEFINE(query8, "$.store.bicycle[1]");
