#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2020-2022, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "generate_input.hpp"
#include "random_distribution_factory.cuh"

#include <cudf/column/column.hpp>
#include <cudf/column/column_factories.hpp>
#include <cudf/detail/gather.hpp>
#include <cudf/detail/valid_if.cuh>
#include <cudf/filling.hpp>
#include <cudf/null_mask.hpp>
#include <cudf/scalar/scalar_factories.hpp>
#include <cudf/table/table.hpp>
#include <cudf/types.hpp>
#include <cudf/utilities/default_stream.hpp>
#include <cudf/utilities/error.hpp>

#include <rmm/device_buffer.hpp>
#include <rmm/device_uvector.hpp>

#include <thrust/binary_search.h>
#include <thrust/copy.h>
#include <thrust/device_ptr.h>
#include <thrust/execution_policy.h>
#include <thrust/fill.h>
#include <thrust/for_each.h>
#include <thrust/functional.h>
#include <thrust/gather.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/iterator/transform_iterator.h>
#include <thrust/iterator/transform_output_iterator.h>
#include <thrust/iterator/zip_iterator.h>
#include <thrust/random/linear_congruential_engine.h>
#include <thrust/random/uniform_int_distribution.h>
#include <thrust/random/uniform_real_distribution.h>
#include <thrust/scan.h>
#include <thrust/tabulate.h>
#include <thrust/transform.h>
#include <thrust/tuple.h>

#include <algorithm>
#include <cstdint>
#include <memory>
#include <optional>
#include <random>
#include <utility>
#include <vector>

/**
 * @brief Mersenne Twister pseudo-random engine.
 */
auto deterministic_engine(unsigned seed) { return thrust::minstd_rand{seed}; }

/**
 *  Computes the mean value for a distribution of given type and value bounds.
 */
template <typename T>
T get_distribution_mean(distribution_params<T> const& dist)
{
  switch (dist.id) {
    case distribution_id::NORMAL:
    case distribution_id::UNIFORM: return (dist.lower_bound / 2.) + (dist.upper_bound / 2.);
    case distribution_id::GEOMETRIC: {
      auto const range_size = dist.lower_bound < dist.upper_bound
                                ? dist.upper_bound - dist.lower_bound
                                : dist.lower_bound - dist.upper_bound;
      auto const p          = geometric_dist_p(range_size);
      if (dist.lower_bound < dist.upper_bound)
        return dist.lower_bound + (1. / p);
      else
        return dist.lower_bound - (1. / p);
    }
    default: CUDF_FAIL("Unsupported distribution type.");
  }
}

/**
 * @brief Computes the average element size in a column, given the data profile.
 *
 * Random distribution parameters like average string length and maximum list nesting level affect
 * the element size of non-fixed-width columns. For lists and structs, `avg_element_size` is called
 * recursively to determine the size of nested columns.
 */
size_t avg_element_size(data_profile const& profile, cudf::data_type dtype);

// Utilities to determine the mean size of an element, given the data profile
template <typename T, CUDF_ENABLE_IF(cudf::is_fixed_width<T>())>
size_t non_fixed_width_size(data_profile const& profile)
{
  CUDF_FAIL("Should not be called, use `size_of` for this type instead");
}

template <typename T, CUDF_ENABLE_IF(!cudf::is_fixed_width<T>())>
size_t non_fixed_width_size(data_profile const& profile)
{
  CUDF_FAIL("not implemented!");
}

template <>
size_t non_fixed_width_size<cudf::string_view>(data_profile const& profile)
{
  auto const dist = profile.get_distribution_params<cudf::string_view>().length_params;
  return get_distribution_mean(dist);
}

template <>
size_t non_fixed_width_size<cudf::list_view>(data_profile const& profile)
{
  auto const dist_params       = profile.get_distribution_params<cudf::list_view>();
  auto const single_level_mean = get_distribution_mean(dist_params.length_params);
  auto const element_size = avg_element_size(profile, cudf::data_type{dist_params.element_type});
  return element_size * pow(single_level_mean, dist_params.max_depth);
}

template <>
size_t non_fixed_width_size<cudf::struct_view>(data_profile const& profile)
{
  auto const dist_params = profile.get_distribution_params<cudf::struct_view>();
  return std::accumulate(dist_params.leaf_types.cbegin(),
                         dist_params.leaf_types.cend(),
                         0ul,
                         [&](auto& sum, auto type_id) {
                           return sum + avg_element_size(profile, cudf::data_type{type_id});
                         });
}

struct non_fixed_width_size_fn {
  template <typename T>
  size_t operator()(data_profile const& profile)
  {
    return non_fixed_width_size<T>(profile);
  }
};

size_t avg_element_size(data_profile const& profile, cudf::data_type dtype)
{
  if (cudf::is_fixed_width(dtype)) { return cudf::size_of(dtype); }
  return cudf::type_dispatcher(dtype, non_fixed_width_size_fn{}, profile);
}

/**
 * @brief bool generator with given probability [0.0 - 1.0] of returning true.
 */
struct bool_generator {
  thrust::minstd_rand engine;
  thrust::uniform_real_distribution<float> dist;
  double probability_true;
  bool_generator(thrust::minstd_rand engine, double probability_true)
    : engine(engine), dist{0, 1}, probability_true{probability_true}
  {
  }
  bool_generator(unsigned seed, double probability_true)
    : engine(seed), dist{0, 1}, probability_true{probability_true}
  {
  }

  __device__ bool operator()(size_t n)
  {
    engine.discard(n);
    return dist(engine) < probability_true;
  }
};

/**
 * @brief Functor that computes a random column element with the given data profile.
 *
 * The implementation is SFINAEd for different type groups. Currently only used for fixed-width
 * types.
 */
template <typename T, typename Enable = void>
struct random_value_fn;

/**
 * @brief Creates an random timestamp/duration value
 */
template <typename T>
struct random_value_fn<T, std::enable_if_t<cudf::is_chrono<T>()>> {
  distribution_fn<int64_t> seconds_gen;
  distribution_fn<int64_t> nanoseconds_gen;

  random_value_fn(distribution_params<T> params)
  {
    using cuda::std::chrono::duration_cast;

    std::pair<cudf::duration_s, cudf::duration_s> const range_s = {
      duration_cast<cuda::std::chrono::seconds>(typename T::duration{params.lower_bound}),
      duration_cast<cuda::std::chrono::seconds>(typename T::duration{params.upper_bound})};
    if (range_s.first != range_s.second) {
      seconds_gen =
        make_distribution<int64_t>(params.id, range_s.first.count(), range_s.second.count());

      nanoseconds_gen = make_distribution<int64_t>(distribution_id::UNIFORM, 0l, 1000000000l);
    } else {
      // Don't need a random seconds generator for sub-second intervals
      seconds_gen = [range_s](thrust::minstd_rand&, size_t size) {
        rmm::device_uvector<int64_t> result(size, cudf::get_default_stream());
        thrust::fill(thrust::device, result.begin(), result.end(), range_s.second.count());
        return result;
      };

      std::pair<cudf::duration_ns, cudf::duration_ns> const range_ns = {
        duration_cast<cudf::duration_ns>(typename T::duration{params.lower_bound}),
        duration_cast<cudf::duration_ns>(typename T::duration{params.upper_bound})};
      nanoseconds_gen = make_distribution<int64_t>(distribution_id::UNIFORM,
                                                   std::min(range_ns.first.count(), 0l),
                                                   std::max(range_ns.second.count(), 0l));
    }
  }

  rmm::device_uvector<T> operator()(thrust::minstd_rand& engine, unsigned size)
  {
    auto const sec = seconds_gen(engine, size);
    auto const ns  = nanoseconds_gen(engine, size);
    rmm::device_uvector<T> result(size, cudf::get_default_stream());
    thrust::transform(
      thrust::device,
      sec.begin(),
      sec.end(),
      ns.begin(),
      result.begin(),
      [] __device__(int64_t sec_value, int64_t nanoseconds_value) {
        auto const timestamp_ns =
          cudf::duration_s{sec_value} + cudf::duration_ns{nanoseconds_value};
        // Return value in the type's precision
        return T(cuda::std::chrono::duration_cast<typename T::duration>(timestamp_ns));
      });
    return result;
  }
};

/**
 * @brief Creates an random fixed_point value.
 */
template <typename T>
struct random_value_fn<T, std::enable_if_t<cudf::is_fixed_point<T>()>> {
  using DeviceType = cudf::device_storage_type_t<T>;
  DeviceType const lower_bound;
  DeviceType const upper_bound;
  distribution_fn<DeviceType> dist;
  std::optional<numeric::scale_type> scale;

  random_value_fn(distribution_params<DeviceType> const& desc)
    : lower_bound{desc.lower_bound},
      upper_bound{desc.upper_bound},
      dist{make_distribution<DeviceType>(desc.id, desc.lower_bound, desc.upper_bound)}
  {
  }

  [[nodiscard]] numeric::scale_type get_scale(thrust::minstd_rand& engine)
  {
    if (not scale.has_value()) {
      constexpr int max_scale = std::numeric_limits<DeviceType>::digits10;
      std::uniform_int_distribution<int> scale_dist{-max_scale, max_scale};
      std::mt19937 engine_scale(engine());
      scale = numeric::scale_type{scale_dist(engine_scale)};
    }
    return scale.value_or(numeric::scale_type{0});
  }

  rmm::device_uvector<DeviceType> operator()(thrust::minstd_rand& engine, unsigned size)
  {
    return dist(engine, size);
  }
};

/**
 * @brief Creates an random numeric value with the given distribution.
 */
template <typename T>
struct random_value_fn<T, std::enable_if_t<!std::is_same_v<T, bool> && cudf::is_numeric<T>()>> {
  T const lower_bound;
  T const upper_bound;
  distribution_fn<T> dist;

  random_value_fn(distribution_params<T> const& desc)
    : lower_bound{desc.lower_bound},
      upper_bound{desc.upper_bound},
      dist{make_distribution<T>(desc.id, desc.lower_bound, desc.upper_bound)}
  {
  }

  auto operator()(thrust::minstd_rand& engine, unsigned size) { return dist(engine, size); }
};

/**
 * @brief Creates an boolean value with given probability of returning `true`.
 */
template <typename T>
struct random_value_fn<T, typename std::enable_if_t<std::is_same_v<T, bool>>> {
  // Bernoulli distribution
  distribution_fn<bool> dist;

  random_value_fn(distribution_params<bool> const& desc)
    : dist{[valid_prob = desc.probability_true](thrust::minstd_rand& engine,
                                                size_t size) -> rmm::device_uvector<bool> {
        rmm::device_uvector<bool> result(size, cudf::get_default_stream());
        thrust::tabulate(
          thrust::device, result.begin(), result.end(), bool_generator(engine, valid_prob));
        return result;
      }}
  {
  }
  auto operator()(thrust::minstd_rand& engine, unsigned size) { return dist(engine, size); }
};

auto create_run_length_dist(cudf::size_type avg_run_len)
{
  // Distribution with low probability of generating 0-1 even with a low `avg_run_len` value
  static constexpr float alpha = 4.f;
  return std::gamma_distribution<float>{alpha, avg_run_len / alpha};
}

/**
 * @brief Generate indices within range [0 , cardinality) repeating with average run length
 * `avg_run_len`
 *
 * @param avg_run_len  Average run length of the generated indices
 * @param cardinality  Number of unique values in the output vector
 * @param num_rows     Number of indices to generate
 * @param engine       Random engine
 * @return Generated indices of type `cudf::size_type`
 */
rmm::device_uvector<cudf::size_type> sample_indices_with_run_length(cudf::size_type avg_run_len,
                                                                    cudf::size_type cardinality,
                                                                    cudf::size_type num_rows,
                                                                    thrust::minstd_rand& engine)
{
  auto sample_dist = random_value_fn<cudf::size_type>{
    distribution_params<cudf::size_type>{distribution_id::UNIFORM, 0, cardinality - 1}};
  if (avg_run_len > 1) {
    auto avglen_dist =
      random_value_fn<int>{distribution_params<int>{distribution_id::UNIFORM, 1, 2 * avg_run_len}};
    auto const approx_run_len = num_rows / avg_run_len + 1;
    auto run_lens             = avglen_dist(engine, approx_run_len);
    thrust::inclusive_scan(
      thrust::device, run_lens.begin(), run_lens.end(), run_lens.begin(), std::plus<int>{});
    auto const samples_indices = sample_dist(engine, approx_run_len + 1);
    // This is gather.
    auto avg_repeated_sample_indices_iterator = thrust::make_transform_iterator(
      thrust::make_counting_iterator(0),
      [rb              = run_lens.begin(),
       re              = run_lens.end(),
       samples_indices = samples_indices.begin()] __device__(cudf::size_type i) {
        auto sample_idx = thrust::upper_bound(thrust::seq, rb, re, i) - rb;
        return samples_indices[sample_idx];
      });
    rmm::device_uvector<cudf::size_type> repeated_sample_indices(num_rows,
                                                                 cudf::get_default_stream());
    thrust::copy(thrust::device,
                 avg_repeated_sample_indices_iterator,
                 avg_repeated_sample_indices_iterator + num_rows,
                 repeated_sample_indices.begin());
    return repeated_sample_indices;
  } else {
    // generate n samples.
    return sample_dist(engine, num_rows);
  }
}

/**
 * @brief Creates a column with random content of type @ref T.
 *
 * @param profile Parameters for the random generator
 * @param engine Pseudo-random engine
 * @param num_rows Size of the output column
 *
 * @tparam T Data type of the output column
 * @return Column filled with random data
 */
template <typename T>
std::unique_ptr<cudf::column> create_random_column(data_profile const& profile,
                                                   thrust::minstd_rand& engine,
                                                   cudf::size_type num_rows)
{
  // Bernoulli distribution
  auto valid_dist = random_value_fn<bool>(
    distribution_params<bool>{1. - profile.get_null_probability().value_or(0)});
  auto value_dist = random_value_fn<T>{profile.get_distribution_params<T>()};

  using DeviceType            = cudf::device_storage_type_t<T>;
  cudf::data_type const dtype = [&]() {
    if constexpr (cudf::is_fixed_point<T>())
      return cudf::data_type{cudf::type_to_id<T>(), value_dist.get_scale(engine)};
    else
      return cudf::data_type{cudf::type_to_id<T>()};
  }();

  // Distribution for picking elements from the array of samples
  auto const avg_run_len = profile.get_avg_run_length();
  rmm::device_uvector<DeviceType> data(0, cudf::get_default_stream());
  rmm::device_uvector<bool> null_mask(0, cudf::get_default_stream());

  if (profile.get_cardinality() == 0 and avg_run_len == 1) {
    data      = value_dist(engine, num_rows);
    null_mask = valid_dist(engine, num_rows);
  } else {
    auto const cardinality = [profile_cardinality = profile.get_cardinality(), num_rows] {
      return (profile_cardinality == 0 or profile_cardinality > num_rows) ? num_rows
                                                                          : profile_cardinality;
    }();
    rmm::device_uvector<bool> samples_null_mask = valid_dist(engine, cardinality);
    rmm::device_uvector<DeviceType> samples     = value_dist(engine, cardinality);

    // generate n samples and gather.
    auto const sample_indices =
      sample_indices_with_run_length(avg_run_len, cardinality, num_rows, engine);
    data      = rmm::device_uvector<DeviceType>(num_rows, cudf::get_default_stream());
    null_mask = rmm::device_uvector<bool>(num_rows, cudf::get_default_stream());
    thrust::gather(
      thrust::device, sample_indices.begin(), sample_indices.end(), samples.begin(), data.begin());
    thrust::gather(thrust::device,
                   sample_indices.begin(),
                   sample_indices.end(),
                   samples_null_mask.begin(),
                   null_mask.begin());
  }

  auto [result_bitmask, null_count] = cudf::detail::valid_if(
    null_mask.begin(), null_mask.end(), thrust::identity<bool>{}, cudf::get_default_stream());

  return std::make_unique<cudf::column>(
    dtype,
    num_rows,
    data.release(),
    profile.get_null_probability().has_value() ? std::move(result_bitmask) : rmm::device_buffer{});
}

struct valid_or_zero {
  template <typename T>
  __device__ T operator()(thrust::tuple<T, bool> len_valid) const
  {
    return thrust::get<1>(len_valid) ? thrust::get<0>(len_valid) : T{0};
  }
};

struct string_generator {
  char* chars;
  thrust::minstd_rand engine;
  thrust::uniform_int_distribution<unsigned char> char_dist;
  string_generator(char* c, thrust::minstd_rand& engine)
    : chars(c), engine(engine), char_dist(32, 137)
  // ~90% ASCII, ~10% UTF-8.
  // ~80% not-space, ~20% space.
  // range 32-127 is ASCII; 127-136 will be multi-byte UTF-8
  {
  }
  __device__ void operator()(thrust::tuple<cudf::size_type, cudf::size_type> str_begin_end)
  {
    auto begin = thrust::get<0>(str_begin_end);
    auto end   = thrust::get<1>(str_begin_end);
    engine.discard(begin);
    for (auto i = begin; i < end; ++i) {
      auto ch = char_dist(engine);
      if (i == end - 1 && ch >= '\x7F') ch = ' ';  // last element ASCII only.
      if (ch >= '\x7F')                            // x7F is at the top edge of ASCII
        chars[i++] = '\xC4';                       // these characters are assigned two bytes
      chars[i] = static_cast<char>(ch + (ch >= '\x7F'));
    }
  }
};

/**
 * @brief Create a UTF-8 string column with the average length.
 *
 */
std::unique_ptr<cudf::column> create_random_utf8_string_column(data_profile const& profile,
                                                               thrust::minstd_rand& engine,
                                                               cudf::size_type num_rows)
{
  auto len_dist =
    random_value_fn<uint32_t>{profile.get_distribution_params<cudf::string_view>().length_params};
  auto valid_dist = random_value_fn<bool>(
    distribution_params<bool>{1. - profile.get_null_probability().value_or(0)});
  auto lengths   = len_dist(engine, num_rows + 1);
  auto null_mask = valid_dist(engine, num_rows + 1);
  thrust::transform_if(
    thrust::device,
    lengths.begin(),
    lengths.end(),
    null_mask.begin(),
    lengths.begin(),
    [] __device__(auto) { return 0; },
    thrust::logical_not<bool>{});
  auto valid_lengths = thrust::make_transform_iterator(
    thrust::make_zip_iterator(thrust::make_tuple(lengths.begin(), null_mask.begin())),
    valid_or_zero{});
  rmm::device_uvector<cudf::size_type> offsets(num_rows + 1, cudf::get_default_stream());
  thrust::exclusive_scan(
    thrust::device, valid_lengths, valid_lengths + lengths.size(), offsets.begin());
  // offfsets are ready.
  auto chars_length = *thrust::device_pointer_cast(offsets.end() - 1);
  rmm::device_uvector<char> chars(chars_length, cudf::get_default_stream());
  thrust::for_each_n(thrust::device,
                     thrust::make_zip_iterator(offsets.begin(), offsets.begin() + 1),
                     num_rows,
                     string_generator{chars.data(), engine});
  auto [result_bitmask, null_count] = cudf::detail::valid_if(
    null_mask.begin(), null_mask.end() - 1, thrust::identity<bool>{}, cudf::get_default_stream());
  return cudf::make_strings_column(
    num_rows,
    std::move(offsets),
    std::move(chars),
    profile.get_null_probability().has_value() ? std::move(result_bitmask) : rmm::device_buffer{});
}

/**
 * @brief Creates a string column with random content.
 *
 * @param profile Parameters for the random generator
 * @param engine Pseudo-random engine
 * @param num_rows Size of the output column
 *
 * @return Column filled with random strings
 */
template <>
std::unique_ptr<cudf::column> create_random_column<cudf::string_view>(data_profile const& profile,
                                                                      thrust::minstd_rand& engine,
                                                                      cudf::size_type num_rows)
{
  auto const cardinality = std::min(profile.get_cardinality(), num_rows);
  auto const avg_run_len = profile.get_avg_run_length();

  auto sample_strings =
    create_random_utf8_string_column(profile, engine, cardinality == 0 ? num_rows : cardinality);
  if (cardinality == 0) { return sample_strings; }
  auto sample_indices = sample_indices_with_run_length(avg_run_len, cardinality, num_rows, engine);
  auto str_table      = cudf::detail::gather(cudf::table_view{{sample_strings->view()}},
                                        sample_indices,
                                        cudf::out_of_bounds_policy::DONT_CHECK,
                                        cudf::detail::negative_index_policy::NOT_ALLOWED,
                                        cudf::get_default_stream());
  return std::move(str_table->release()[0]);
}

template <>
std::unique_ptr<cudf::column> create_random_column<cudf::dictionary32>(data_profile const& profile,
                                                                       thrust::minstd_rand& engine,
                                                                       cudf::size_type num_rows)
{
  CUDF_FAIL("not implemented yet");
}

/**
 * @brief Functor to dispatch create_random_column calls.
 */
struct create_rand_col_fn {
 public:
  template <typename T>
  std::unique_ptr<cudf::column> operator()(data_profile const& profile,
                                           thrust::minstd_rand& engine,
                                           cudf::size_type num_rows)
  {
    return create_random_column<T>(profile, engine, num_rows);
  }
};

/**
 * @brief Calculates the number of direct parents needed to generate a struct column hierarchy with
 * lowest maximum number of children in any nested column.
 *
 * Used to generate an "evenly distributed" struct column hierarchy with the given number of leaf
 * columns and nesting levels. The column tree is considered evenly distributed if all columns have
 * nearly the same number of child columns (difference not larger than one).
 */
int num_direct_parents(int num_lvls, int num_leaf_columns)
{
  // Estimated average number of children in the hierarchy;
  auto const num_children_avg = std::pow(num_leaf_columns, 1. / num_lvls);
  // Minimum number of children columns for any column in the hierarchy
  int const num_children_min = std::floor(num_children_avg);
  // Maximum number of children columns for any column in the hierarchy
  int const num_children_max = num_children_min + 1;

  // Minimum number of columns needed so that their number of children does not exceed the maximum
  int const min_for_current_nesting = std::ceil((double)num_leaf_columns / num_children_max);
  // Minimum number of columns needed so that columns at the higher levels have at least the minimum
  // number of children
  int const min_for_upper_nesting = std::pow(num_children_min, num_lvls - 1);
  // Both conditions need to be satisfied
  return std::max(min_for_current_nesting, min_for_upper_nesting);
}

template <>
std::unique_ptr<cudf::column> create_random_column<cudf::struct_view>(data_profile const& profile,
                                                                      thrust::minstd_rand& engine,
                                                                      cudf::size_type num_rows)
{
  auto const dist_params = profile.get_distribution_params<cudf::struct_view>();

  // Generate leaf columns
  std::vector<std::unique_ptr<cudf::column>> children;
  children.reserve(dist_params.leaf_types.size());
  std::transform(dist_params.leaf_types.cbegin(),
                 dist_params.leaf_types.cend(),
                 std::back_inserter(children),
                 [&](auto& type_id) {
                   return cudf::type_dispatcher(
                     cudf::data_type(type_id), create_rand_col_fn{}, profile, engine, num_rows);
                 });

  auto valid_dist = random_value_fn<bool>(
    distribution_params<bool>{1. - profile.get_null_probability().value_or(0)});

  // Generate the column bottom-up
  for (int lvl = dist_params.max_depth; lvl > 0; --lvl) {
    // Generating the next level
    std::vector<std::unique_ptr<cudf::column>> parents;
    parents.resize(num_direct_parents(lvl, children.size()));

    auto current_child = children.begin();
    for (auto current_parent = parents.begin(); current_parent != parents.end(); ++current_parent) {
      auto [null_mask, null_count] = [&]() {
        if (profile.get_null_probability().has_value()) {
          auto valids = valid_dist(engine, num_rows);
          return cudf::detail::valid_if(
            valids.begin(), valids.end(), thrust::identity<bool>{}, cudf::get_default_stream());
        }
        return std::pair<rmm::device_buffer, cudf::size_type>{};
      }();

      // Adopt remaining children as evenly as possible
      auto const num_to_adopt = cudf::util::div_rounding_up_unsafe(
        std::distance(current_child, children.end()), std::distance(current_parent, parents.end()));
      CUDF_EXPECTS(num_to_adopt > 0, "No children columns left to adopt");

      std::vector<std::unique_ptr<cudf::column>> children_to_adopt;
      children_to_adopt.insert(children_to_adopt.end(),
                               std::make_move_iterator(current_child),
                               std::make_move_iterator(current_child + num_to_adopt));
      current_child += children_to_adopt.size();

      *current_parent = cudf::make_structs_column(
        num_rows, std::move(children_to_adopt), null_count, std::move(null_mask));
    }

    if (lvl == 1) {
      CUDF_EXPECTS(parents.size() == 1, "There should be one top-level column");
      return std::move(parents.front());
    }
    children = std::move(parents);
  }
  CUDF_FAIL("Reached unreachable code in struct column creation");
}

template <typename T>
struct clamp_down : public thrust::unary_function<T, T> {
  T max;
  clamp_down(T max) : max(max) {}
  __host__ __device__ T operator()(T x) const { return min(x, max); }
};
/**
 * @brief Creates a list column with random content.
 *
 * The data profile determines the list length distribution, number of nested level, and the data
 * type of the bottom level.
 *
 * @param profile Parameters for the random generator
 * @param engine Pseudo-random engine
 * @param num_rows Size of the output column
 *
 * @return Column filled with random lists
 */
template <>
std::unique_ptr<cudf::column> create_random_column<cudf::list_view>(data_profile const& profile,
                                                                    thrust::minstd_rand& engine,
                                                                    cudf::size_type num_rows)
{
  auto const dist_params       = profile.get_distribution_params<cudf::list_view>();
  auto const single_level_mean = get_distribution_mean(dist_params.length_params);
  auto const num_elements      = num_rows * pow(single_level_mean, dist_params.max_depth);

  auto leaf_column = cudf::type_dispatcher(
    cudf::data_type(dist_params.element_type), create_rand_col_fn{}, profile, engine, num_elements);
  auto len_dist =
    random_value_fn<uint32_t>{profile.get_distribution_params<cudf::list_view>().length_params};
  auto valid_dist = random_value_fn<bool>(
    distribution_params<bool>{1. - profile.get_null_probability().value_or(0)});

  // Generate the list column bottom-up
  auto list_column = std::move(leaf_column);
  for (int lvl = 0; lvl < dist_params.max_depth; ++lvl) {
    // Generating the next level - offsets point into the current list column
    auto current_child_column      = std::move(list_column);
    cudf::size_type const num_rows = current_child_column->size() / single_level_mean;

    auto offsets = len_dist(engine, num_rows + 1);
    auto valids  = valid_dist(engine, num_rows);
    // to ensure these values <= current_child_column->size()
    auto output_offsets = thrust::make_transform_output_iterator(
      offsets.begin(), clamp_down{current_child_column->size()});

    thrust::exclusive_scan(thrust::device, offsets.begin(), offsets.end(), output_offsets);
    thrust::device_pointer_cast(offsets.end())[-1] =
      current_child_column->size();  // Always include all elements

    auto offsets_column = std::make_unique<cudf::column>(
      cudf::data_type{cudf::type_id::INT32}, num_rows + 1, offsets.release());

    auto [null_mask, null_count] = cudf::detail::valid_if(
      valids.begin(), valids.end(), thrust::identity<bool>{}, cudf::get_default_stream());
    list_column = cudf::make_lists_column(
      num_rows,
      std::move(offsets_column),
      std::move(current_child_column),
      profile.get_null_probability().has_value() ? null_count : 0,  // cudf::UNKNOWN_NULL_COUNT,
      profile.get_null_probability().has_value() ? std::move(null_mask) : rmm::device_buffer{});
  }
  return list_column;  // return the top-level column
}

using columns_vector = std::vector<std::unique_ptr<cudf::column>>;

/**
 * @brief Creates a vector of columns with random content.
 *
 * @param profile Parameters for the random generator
 * @param dtype_ids vector of data type ids, one for each output column
 * @param engine Pseudo-random engine
 * @param num_rows Size of the output columns
 *
 * @return Column filled with random lists
 */
columns_vector create_random_columns(data_profile const& profile,
                                     std::vector<cudf::type_id> dtype_ids,
                                     thrust::minstd_rand engine,
                                     cudf::size_type num_rows)
{
  columns_vector output_columns;
  std::transform(
    dtype_ids.begin(), dtype_ids.end(), std::back_inserter(output_columns), [&](auto tid) {
      engine.discard(num_rows);
      return cudf::type_dispatcher(
        cudf::data_type(tid), create_rand_col_fn{}, profile, engine, num_rows);
    });
  return output_columns;
}

/**
 * @brief Repeats the input data types cyclically order to fill a vector of @ref num_cols
 * elements.
 */
std::vector<cudf::type_id> cycle_dtypes(std::vector<cudf::type_id> const& dtype_ids,
                                        cudf::size_type num_cols)
{
  if (dtype_ids.size() == static_cast<std::size_t>(num_cols)) { return dtype_ids; }
  std::vector<cudf::type_id> out_dtypes;
  out_dtypes.reserve(num_cols);
  for (cudf::size_type col = 0; col < num_cols; ++col)
    out_dtypes.push_back(dtype_ids[col % dtype_ids.size()]);
  return out_dtypes;
}

std::unique_ptr<cudf::table> create_random_table(std::vector<cudf::type_id> const& dtype_ids,
                                                 table_size_bytes table_bytes,
                                                 data_profile const& profile,
                                                 unsigned seed)
{
  size_t const avg_row_bytes =
    std::accumulate(dtype_ids.begin(), dtype_ids.end(), 0ul, [&](size_t sum, auto tid) {
      return sum + avg_element_size(profile, cudf::data_type(tid));
    });
  cudf::size_type const num_rows = table_bytes.size / avg_row_bytes;

  return create_random_table(dtype_ids, row_count{num_rows}, profile, seed);
}

std::unique_ptr<cudf::table> create_random_table(std::vector<cudf::type_id> const& dtype_ids,
                                                 row_count num_rows,
                                                 data_profile const& profile,
                                                 unsigned seed)
{
  auto seed_engine = deterministic_engine(seed);
  thrust::uniform_int_distribution<unsigned> seed_dist;

  columns_vector output_columns;
  std::transform(
    dtype_ids.begin(), dtype_ids.end(), std::back_inserter(output_columns), [&](auto tid) mutable {
      return create_random_column(tid, num_rows, profile, seed_dist(seed_engine));
    });
  return std::make_unique<cudf::table>(std::move(output_columns));
}

std::unique_ptr<cudf::column> create_random_column(cudf::type_id dtype_id,
                                                   row_count num_rows,
                                                   data_profile const& profile,
                                                   unsigned seed)
{
  auto engine = deterministic_engine(seed);
  return cudf::type_dispatcher(
    cudf::data_type(dtype_id), create_rand_col_fn{}, profile, engine, num_rows.count);
}

std::unique_ptr<cudf::table> create_sequence_table(std::vector<cudf::type_id> const& dtype_ids,
                                                   row_count num_rows,
                                                   std::optional<double> null_probability,
                                                   unsigned seed)
{
  auto seed_engine = deterministic_engine(seed);
  thrust::uniform_int_distribution<unsigned> seed_dist;

  auto columns = std::vector<std::unique_ptr<cudf::column>>(dtype_ids.size());
  std::transform(dtype_ids.begin(), dtype_ids.end(), columns.begin(), [&](auto dtype) mutable {
    auto init = cudf::make_default_constructed_scalar(cudf::data_type{dtype});
    auto col  = cudf::sequence(num_rows.count, *init);
    auto [mask, count] =
      create_random_null_mask(num_rows.count, null_probability, seed_dist(seed_engine));
    col->set_null_mask(std::move(mask), count);
    return col;
  });
  return std::make_unique<cudf::table>(std::move(columns));
}

std::pair<rmm::device_buffer, cudf::size_type> create_random_null_mask(
  cudf::size_type size, std::optional<double> null_probability, unsigned seed)
{
  if (not null_probability.has_value()) { return {rmm::device_buffer{}, 0}; }
  CUDF_EXPECTS(*null_probability >= 0.0 and *null_probability <= 1.0,
               "Null probability must be within the range [0.0, 1.0]");
  if (*null_probability == 0.0f) {
    return {cudf::create_null_mask(size, cudf::mask_state::ALL_VALID), 0};
  } else if (*null_probability == 1.0) {
    return {cudf::create_null_mask(size, cudf::mask_state::ALL_NULL), size};
  } else {
    return cudf::detail::valid_if(thrust::make_counting_iterator<cudf::size_type>(0),
                                  thrust::make_counting_iterator<cudf::size_type>(size),
                                  bool_generator{seed, 1.0 - *null_probability},
                                  cudf::get_default_stream());
  }
}

std::vector<cudf::type_id> get_type_or_group(int32_t id)
{
  // identity transformation when passing a concrete type_id
  if (id < static_cast<int32_t>(cudf::type_id::NUM_TYPE_IDS))
    return {static_cast<cudf::type_id>(id)};

  // if the value is larger that type_id::NUM_TYPE_IDS, it's a group id
  type_group_id const group_id = static_cast<type_group_id>(id);

  using trait_fn       = bool (*)(cudf::data_type);
  trait_fn is_integral = [](cudf::data_type type) {
    return cudf::is_numeric(type) && !cudf::is_floating_point(type);
  };
  trait_fn is_integral_signed = [](cudf::data_type type) {
    return cudf::is_numeric(type) && !cudf::is_floating_point(type) && !cudf::is_unsigned(type);
  };
  auto fn = [&]() -> trait_fn {
    switch (group_id) {
      case type_group_id::FLOATING_POINT: return cudf::is_floating_point;
      case type_group_id::INTEGRAL: return is_integral;
      case type_group_id::INTEGRAL_SIGNED: return is_integral_signed;
      case type_group_id::NUMERIC: return cudf::is_numeric;
      case type_group_id::TIMESTAMP: return cudf::is_timestamp;
      case type_group_id::DURATION: return cudf::is_duration;
      case type_group_id::FIXED_POINT: return cudf::is_fixed_point;
      case type_group_id::COMPOUND: return cudf::is_compound;
      case type_group_id::NESTED: return cudf::is_nested;
      default: CUDF_FAIL("Invalid data type group");
    }
  }();
  std::vector<cudf::type_id> types;
  for (int type_int = 0; type_int < static_cast<int32_t>(cudf::type_id::NUM_TYPE_IDS); ++type_int) {
    auto const type = static_cast<cudf::type_id>(type_int);
    if (type != cudf::type_id::EMPTY && fn(cudf::data_type(type))) types.push_back(type);
  }
  return types;
}

std::vector<cudf::type_id> get_type_or_group(std::vector<int32_t> const& ids)
{
  std::vector<cudf::type_id> all_type_ids;
  for (auto& id : ids) {
    auto const type_ids = get_type_or_group(id);
    all_type_ids.insert(std::end(all_type_ids), std::cbegin(type_ids), std::cend(type_ids));
  }
  return all_type_ids;
}
