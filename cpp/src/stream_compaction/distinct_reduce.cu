#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2022, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "distinct_reduce.cuh"

#include <thrust/for_each.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/uninitialized_fill.h>

namespace cudf::detail {

namespace {
/**
 * @brief A functor to perform reduce-by-key with keys are rows that compared equal.
 *
 * TODO: We need to switch to use `static_reduction_map` when it is ready
 * (https://github.com/NVIDIA/cuCollections/pull/98).
 */
template <typename MapView, typename KeyHasher, typename KeyEqual>
struct reduce_by_row_fn {
  MapView const d_map;
  KeyHasher const d_hasher;
  KeyEqual const d_equal;
  duplicate_keep_option const keep;
  size_type* const d_output;

  reduce_by_row_fn(MapView const& d_map,
                   KeyHasher const& d_hasher,
                   KeyEqual const& d_equal,
                   duplicate_keep_option const keep,
                   size_type* const d_output)
    : d_map{d_map}, d_hasher{d_hasher}, d_equal{d_equal}, keep{keep}, d_output{d_output}
  {
  }

  __device__ void operator()(size_type const idx) const
  {
    auto const out_ptr = get_output_ptr(idx);

    if (keep == duplicate_keep_option::KEEP_FIRST) {
      // Store the smallest index of all rows that are equal.
      atomicMin(out_ptr, idx);
    } else if (keep == duplicate_keep_option::KEEP_LAST) {
      // Store the greatest index of all rows that are equal.
      atomicMax(out_ptr, idx);
    } else {
      // Count the number of rows in each group of rows that are compared equal.
      atomicAdd(out_ptr, size_type{1});
    }
  }

 private:
  __device__ size_type* get_output_ptr(size_type const idx) const
  {
    auto const iter = d_map.find(idx, d_hasher, d_equal);

    if (iter != d_map.end()) {
      // Only one index value of the duplicate rows could be inserted into the map.
      // As such, looking up for all indices of duplicate rows always returns the same value.
      auto const inserted_idx = iter->second.load(cuda::std::memory_order_relaxed);

      // All duplicate rows will have concurrent access to this same output slot.
      return &d_output[inserted_idx];
    } else {
      // All input `idx` values have been inserted into the map before.
      // Thus, searching for an `idx` key resulting in the `end()` iterator only happens if
      // `d_equal(idx, idx) == false`.
      // Such situations are due to comparing nulls or NaNs which are considered as always unequal.
      // In those cases, all rows containing nulls or NaNs are distinct. Just return their direct
      // output slot.
      return &d_output[idx];
    }
  }
};

}  // namespace

rmm::device_uvector<size_type> hash_reduce_by_row(
  hash_map_type const& map,
  std::shared_ptr<cudf::experimental::row::equality::preprocessed_table> const preprocessed_input,
  size_type num_rows,
  cudf::nullate::DYNAMIC has_nulls,
  duplicate_keep_option keep,
  null_equality nulls_equal,
  nan_equality nans_equal,
  rmm::cuda_stream_view stream,
  rmm::mr::device_memory_resource* mr)
{
  CUDF_EXPECTS(keep != duplicate_keep_option::KEEP_ANY,
               "This function should not be called with KEEP_ANY");

  auto reduction_results = rmm::device_uvector<size_type>(num_rows, stream, mr);

  thrust::uninitialized_fill(rmm::exec_policy(stream),
                             reduction_results.begin(),
                             reduction_results.end(),
                             reduction_init_value(keep));

  auto const row_hasher = cudf::experimental::row::hash::row_hasher(preprocessed_input);
  auto const key_hasher = experimental::compaction_hash(row_hasher.device_hasher(has_nulls));

  auto const row_comp = cudf::experimental::row::equality::self_comparator(preprocessed_input);

  auto const reduce_by_row = [&](auto const value_comp) {
    auto const key_equal = row_comp.equal_to(has_nulls, nulls_equal, value_comp);
    thrust::for_each(
      rmm::exec_policy(stream),
      thrust::make_counting_iterator(0),
      thrust::make_counting_iterator(num_rows),
      reduce_by_row_fn{
        map.get_device_view(), key_hasher, key_equal, keep, reduction_results.begin()});
  };

  if (nans_equal == nan_equality::ALL_EQUAL) {
    using nan_equal_comparator =
      cudf::experimental::row::equality::nan_equal_physical_equality_comparator;
    reduce_by_row(nan_equal_comparator{});
  } else {
    using nan_unequal_comparator = cudf::experimental::row::equality::physical_equality_comparator;
    reduce_by_row(nan_unequal_comparator{});
  }

  return reduction_results;
}

}  // namespace cudf::detail
