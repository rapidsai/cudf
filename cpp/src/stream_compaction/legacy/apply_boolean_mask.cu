/*
 * Copyright (c) 2019, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <cudf/legacy/table.hpp>
#include "copy_if.cuh"

namespace {
// Returns true if the mask is true and valid (non-null) for index i
// This is the filter functor for apply_boolean_mask
// Note we use a functor here so we can cast to a bitmask_t __restrict__
// pointer on the host side, which we can't do with a lambda.
template <bool has_data, bool has_nulls>
struct boolean_mask_filter {
  boolean_mask_filter(gdf_column const &boolean_mask)
    : data{static_cast<bool *>(boolean_mask.data)},
      bitmask{reinterpret_cast<bit_mask_t *>(boolean_mask.valid)}
  {
  }

  __device__ inline bool operator()(cudf::size_type i)
  {
    bool valid   = !has_nulls || bit_mask::is_valid(bitmask, i);
    bool is_true = !has_data || data[i];
    return is_true && valid;
  }

  bool const *__restrict__ data;
  bit_mask_t const *__restrict__ bitmask;
};

}  // namespace

namespace cudf {
/*
 * Filters a table using a column of boolean values as a mask.
 *
 * calls copy_if() with the `boolean_mask_filter` functor.
 */
table apply_boolean_mask(table const &input, gdf_column const &boolean_mask)
{
  if (boolean_mask.size == 0) return empty_like(input);

  CUDF_EXPECTS(boolean_mask.dtype == GDF_BOOL8, "Mask must be Boolean type");
  CUDF_EXPECTS(boolean_mask.data != nullptr || boolean_mask.valid != nullptr, "Null boolean_mask");
  // zero-size inputs are OK, but otherwise input size must match mask size
  CUDF_EXPECTS(input.num_rows() == 0 || input.num_rows() == boolean_mask.size,
               "Column size mismatch");

  if (boolean_mask.data == nullptr)
    return detail::copy_if(input, boolean_mask_filter<false, true>{boolean_mask});
  else if (not cudf::has_nulls(boolean_mask))
    return detail::copy_if(input, boolean_mask_filter<true, false>{boolean_mask});
  else
    return detail::copy_if(input, boolean_mask_filter<true, true>{boolean_mask});
}

}  // namespace cudf
