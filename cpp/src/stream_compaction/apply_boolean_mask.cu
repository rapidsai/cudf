/*
 * Copyright (c) 2019, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <cudf/column/column_device_view.cuh>
#include <cudf/column/column_view.hpp>
#include <cudf/detail/copy_if.cuh>
#include <cudf/detail/nvtx/ranges.hpp>
#include <cudf/detail/stream_compaction.hpp>
#include <cudf/stream_compaction.hpp>
#include <cudf/table/table.hpp>
#include <cudf/table/table_view.hpp>
#include <cudf/types.hpp>
#include <cudf/utilities/type_dispatcher.hpp>

#include <algorithm>

namespace {
// Returns true if the mask is true and valid (non-null) for index i
// This is the filter functor for apply_boolean_mask
template <bool has_nulls = true>
struct boolean_mask_filter {
  boolean_mask_filter(cudf::column_device_view const& boolean_mask) : boolean_mask{boolean_mask} {}

  __device__ inline bool operator()(cudf::size_type i)
  {
    if (true == has_nulls) {
      bool valid   = boolean_mask.is_valid(i);
      bool is_true = boolean_mask.data<bool>()[i];

      return is_true && valid;
    } else {
      return boolean_mask.data<bool>()[i];
    }
  }

 protected:
  cudf::column_device_view boolean_mask;
};

}  // namespace

namespace cudf {
namespace detail {
/*
 * Filters a table_view using a column_view of boolean values as a mask.
 *
 * calls copy_if() with the `boolean_mask_filter` functor.
 */
std::unique_ptr<table> apply_boolean_mask(table_view const& input,
                                          column_view const& boolean_mask,
                                          rmm::mr::device_memory_resource* mr,
                                          hipStream_t stream)
{
  if (boolean_mask.is_empty()) { return empty_like(input); }

  CUDF_EXPECTS(boolean_mask.type().id() == type_id::BOOL8, "Mask must be Boolean type");
  // zero-size inputs are OK, but otherwise input size must match mask size
  CUDF_EXPECTS(input.num_rows() == 0 || input.num_rows() == boolean_mask.size(),
               "Column size mismatch");

  auto device_boolean_mask = cudf::column_device_view::create(boolean_mask, stream);

  if (boolean_mask.has_nulls()) {
    return detail::copy_if(input, boolean_mask_filter<true>{*device_boolean_mask}, mr, stream);
  } else {
    return detail::copy_if(input, boolean_mask_filter<false>{*device_boolean_mask}, mr, stream);
  }
}

}  // namespace detail

/*
 * Filters a table_view using a column_view of boolean values as a mask.
 */
std::unique_ptr<table> apply_boolean_mask(table_view const& input,
                                          column_view const& boolean_mask,
                                          rmm::mr::device_memory_resource* mr)
{
  CUDF_FUNC_RANGE();
  return detail::apply_boolean_mask(input, boolean_mask, mr);
}
}  // namespace cudf
