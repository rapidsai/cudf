#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2022, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "stream_compaction_common.cuh"
#include "stream_compaction_common.hpp"

#include <cudf/column/column_device_view.cuh>
#include <cudf/column/column_factories.hpp>
#include <cudf/column/column_view.hpp>
#include <cudf/detail/iterator.cuh>
#include <cudf/detail/null_mask.hpp>
#include <cudf/detail/nvtx/ranges.hpp>
#include <cudf/detail/sorting.hpp>
#include <cudf/detail/stream_compaction.hpp>
#include <cudf/stream_compaction.hpp>
#include <cudf/table/table_view.hpp>
#include <cudf/utilities/type_dispatcher.hpp>

#include <rmm/cuda_stream_view.hpp>
#include <rmm/exec_policy.hpp>

#include <thrust/count.h>
#include <thrust/execution_policy.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/logical.h>

#include <cmath>
#include <cstddef>
#include <type_traits>
#include <utility>
#include <vector>

namespace cudf {
namespace detail {
namespace {
/**
 * @brief A functor to be used along with device type_dispatcher to check if
 * the row `index` of `column_device_view` is `NaN`.
 */
struct check_nan {
  // Check if a value is `NaN` for floating point type columns
  template <typename T, std::enable_if_t<std::is_floating_point_v<T>>* = nullptr>
  __device__ inline bool operator()(column_device_view const& input, size_type index)
  {
    return std::isnan(input.data<T>()[index]);
  }
  // Non-floating point type columns can never have `NaN`, so it will always return false.
  template <typename T, std::enable_if_t<not std::is_floating_point_v<T>>* = nullptr>
  __device__ inline bool operator()(column_device_view const&, size_type)
  {
    return false;
  }
};
}  // namespace

cudf::size_type unique_count(table_view const& keys,
                             null_equality nulls_equal,
                             rmm::cuda_stream_view stream)
{
  auto table_ptr = cudf::table_device_view::create(keys, stream);
  row_equality_comparator comp(
    nullate::DYNAMIC{cudf::has_nulls(keys)}, *table_ptr, *table_ptr, nulls_equal);
  return thrust::count_if(
    rmm::exec_policy(stream),
    thrust::counting_iterator<cudf::size_type>(0),
    thrust::counting_iterator<cudf::size_type>(keys.num_rows()),
    [comp] __device__(cudf::size_type i) { return (i == 0 or not comp(i, i - 1)); });
}

cudf::size_type unique_count(column_view const& input,
                             null_policy null_handling,
                             nan_policy nan_handling,
                             rmm::cuda_stream_view stream)
{
  auto const num_rows = input.size();

  if (num_rows == 0 or num_rows == input.null_count()) { return 0; }

  auto const count_nulls      = null_handling == null_policy::INCLUDE;
  auto const nan_is_null      = nan_handling == nan_policy::NAN_IS_NULL;
  auto const should_check_nan = cudf::is_floating_point(input.type());
  auto input_device_view      = cudf::column_device_view::create(input, stream);
  auto device_view            = *input_device_view;
  auto input_table_view       = table_view{{input}};
  auto table_ptr              = cudf::table_device_view::create(input_table_view, stream);
  row_equality_comparator comp(nullate::DYNAMIC{cudf::has_nulls(input_table_view)},
                               *table_ptr,
                               *table_ptr,
                               null_equality::EQUAL);

  return thrust::count_if(
    rmm::exec_policy(stream),
    thrust::counting_iterator<cudf::size_type>(0),
    thrust::counting_iterator<cudf::size_type>(num_rows),
    [count_nulls, nan_is_null, should_check_nan, device_view, comp] __device__(cudf::size_type i) {
      auto const is_null = device_view.is_null(i);
      auto const is_nan  = nan_is_null and should_check_nan and
                          cudf::type_dispatcher(device_view.type(), check_nan{}, device_view, i);
      if (not count_nulls and (is_null or (nan_is_null and is_nan))) { return false; }
      if (i == 0) { return true; }
      if (count_nulls and nan_is_null and (is_nan or is_null)) {
        auto const prev_is_nan =
          should_check_nan and
          cudf::type_dispatcher(device_view.type(), check_nan{}, device_view, i - 1);
        return not(prev_is_nan or device_view.is_null(i - 1));
      }
      return not comp(i, i - 1);
    });
}
}  // namespace detail

cudf::size_type unique_count(column_view const& input,
                             null_policy null_handling,
                             nan_policy nan_handling)
{
  CUDF_FUNC_RANGE();
  return detail::unique_count(input, null_handling, nan_handling, cudf::get_default_stream());
}

cudf::size_type unique_count(table_view const& input, null_equality nulls_equal)
{
  CUDF_FUNC_RANGE();
  return detail::unique_count(input, nulls_equal);
}

}  // namespace cudf
