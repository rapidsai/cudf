#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2022-2024, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "distinct_helpers.hpp"

#include <cudf/detail/hash_reduce_by_row.cuh>

#include <rmm/resource_ref.hpp>

namespace cudf::detail {

namespace {
/**
 * @brief The functor to find the first/last/all duplicate row for rows that compared equal.
 */
template <typename MapView, typename KeyHasher, typename KeyEqual>
struct reduce_fn : reduce_by_row_fn_base<MapView, KeyHasher, KeyEqual, size_type> {
  duplicate_keep_option const keep;

  reduce_fn(MapView const& d_map,
            KeyHasher const& d_hasher,
            KeyEqual const& d_equal,
            duplicate_keep_option const keep,
            size_type* const d_output)
    : reduce_by_row_fn_base<MapView, KeyHasher, KeyEqual, size_type>{d_map,
                                                                     d_hasher,
                                                                     d_equal,
                                                                     d_output},
      keep{keep}
  {
  }

  __device__ void operator()(size_type const idx) const
  {
    auto const out_ptr = this->get_output_ptr(idx);

    if (keep == duplicate_keep_option::KEEP_FIRST) {
      // Store the smallest index of all rows that are equal.
      atomicMin(out_ptr, idx);
    } else if (keep == duplicate_keep_option::KEEP_LAST) {
      // Store the greatest index of all rows that are equal.
      atomicMax(out_ptr, idx);
    } else {
      // Count the number of rows in each group of rows that are compared equal.
      atomicAdd(out_ptr, size_type{1});
    }
  }
};

/**
 * @brief The builder to construct an instance of `reduce_fn` functor base on the given
 * value of the `duplicate_keep_option` member variable.
 */
struct reduce_func_builder {
  duplicate_keep_option const keep;

  template <typename MapView, typename KeyHasher, typename KeyEqual>
  auto build(MapView const& d_map,
             KeyHasher const& d_hasher,
             KeyEqual const& d_equal,
             size_type* const d_output)
  {
    return reduce_fn<MapView, KeyHasher, KeyEqual>{d_map, d_hasher, d_equal, keep, d_output};
  }
};

}  // namespace

// This function is split from `distinct.cu` to improve compile time.
rmm::device_uvector<size_type> reduce_by_row(
  hash_map_type const& map,
  std::shared_ptr<cudf::experimental::row::equality::preprocessed_table> const preprocessed_input,
  size_type num_rows,
  cudf::nullate::DYNAMIC has_nulls,
  bool has_nested_columns,
  duplicate_keep_option keep,
  null_equality nulls_equal,
  nan_equality nans_equal,
  rmm::cuda_stream_view stream,
  rmm::device_async_resource_ref mr)
{
  CUDF_EXPECTS(keep != duplicate_keep_option::KEEP_ANY,
               "This function should not be called with KEEP_ANY");

  return hash_reduce_by_row(map,
                            preprocessed_input,
                            num_rows,
                            has_nulls,
                            has_nested_columns,
                            nulls_equal,
                            nans_equal,
                            reduce_func_builder{keep},
                            reduction_init_value(keep),
                            stream,
                            mr);
}

}  // namespace cudf::detail
