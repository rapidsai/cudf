/*
 * Copyright (c) 2022, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <cudf/detail/copy_if.cuh>
#include <cudf/detail/stream_compaction.hpp>
#include <cudf/table/table.hpp>
#include <cudf/table/table_view.hpp>
#include <cudf/types.hpp>

#include <thrust/iterator/constant_iterator.h>
#include <thrust/scatter.h>
#include <thrust/uninitialized_fill.h>

namespace cudf::detail {

std::unique_ptr<table> stable_distinct(table_view const& input,
                                       std::vector<size_type> const& keys,
                                       duplicate_keep_option keep,
                                       null_equality nulls_equal,
                                       nan_equality nans_equal,
                                       rmm::cuda_stream_view stream,
                                       rmm::mr::device_memory_resource* mr)
{
  if (input.num_rows() == 0 or input.num_columns() == 0 or keys.empty()) {
    return empty_like(input);
  }

  auto const distinct_indices =
    get_distinct_indices(input.select(keys), keep, nulls_equal, nans_equal, stream);

  // Markers to denote which rows to be copied to the output.
  auto const output_markers = [&] {
    auto markers = rmm::device_uvector<bool>(input.num_rows(), stream);
    thrust::uninitialized_fill(rmm::exec_policy(stream), markers.begin(), markers.end(), false);
    thrust::scatter(
      rmm::exec_policy(stream),
      thrust::constant_iterator<bool>(true, 0),
      thrust::constant_iterator<bool>(true, static_cast<size_type>(distinct_indices.size())),
      distinct_indices.begin(),
      markers.begin());
    return markers;
  }();

  return cudf::detail::copy_if(
    input,
    [output_markers = output_markers.begin()] __device__(auto const idx) {
      return *(output_markers + idx);
    },
    stream,
    mr);
}

}  // namespace cudf::detail
