/*
 * Copyright (c) 2020, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <algorithm>
#include <cstdint>
#include <cudf/strings/detail/utilities.hpp>
#include <memory>

#include "cudf/column/column_device_view.cuh"
#include "cudf/detail/nvtx/ranges.hpp"
#include "cudf/replace.hpp"
#include "cudf/strings/detail/utilities.cuh"
#include "cudf/types.hpp"
#include "cudf/utilities/traits.hpp"
#include "cudf/utilities/type_dispatcher.hpp"

namespace cudf {
namespace detail {
struct ByteListConversion {
  /**
   * @brief Function object for converting primitive types and string columns to lists of bytes,
   * mimics Spark's cast to binary type.
   */
  struct flip_endianness {
    char* d_chars;
    const char* d_data;
    uint32_t mask;
    __device__ void operator()(int byte_index)
    {
      d_chars[byte_index] = d_data[byte_index + mask - ((byte_index & mask) << 1)];
    }
  };

  template <
    typename T,
    typename std::enable_if_t<!std::is_integral<T>::value and !is_floating_point<T>()>* = nullptr>
  std::unique_ptr<column> operator()(column_view const& input_column,
                                     endianess_policy configuration,
                                     rmm::mr::device_memory_resource* mr,
                                     hipStream_t stream) const
  {
    release_assert(false && "MD5 Unsupported non-numeric and non-string column");
    return NULL;
  }

  template <typename T, typename std::enable_if_t<is_floating_point<T>()>* = nullptr>
  std::unique_ptr<column> operator()(column_view const& input_column,
                                     endianess_policy configuration,
                                     rmm::mr::device_memory_resource* mr,
                                     hipStream_t stream) const
  {
    size_type num_output_elements = input_column.size() * cudf::size_of(input_column.type());

    auto begin          = thrust::make_constant_iterator(cudf::size_of(input_column.type()));
    auto offsets_column = cudf::strings::detail::make_offsets_child_column(
      begin, begin + input_column.size(), mr, stream);

    auto byte_column = make_numeric_column(
      data_type{type_id::UINT8}, num_output_elements, mask_state::UNALLOCATED, stream, mr);
    auto d_chars = byte_column->mutable_view().data<char>();

    rmm::device_buffer null_mask = copy_bitmask(input_column, stream, mr);
    auto normalized              = normalize_nans_and_zeros(input_column);

    if (configuration == endianess_policy::FLIP_ENDIANESS) {
      uint32_t mask = cudf::size_of(input_column.type()) - 1;
      thrust::for_each(rmm::exec_policy(stream)->on(stream),
                       thrust::make_counting_iterator(0),
                       thrust::make_counting_iterator(num_output_elements),
                       flip_endianness{d_chars, normalized->view().data<char>(), mask});
    } else {
      thrust::copy_n(rmm::exec_policy(stream)->on(stream),
                     normalized->view().data<char>(),
                     num_output_elements,
                     d_chars);
    }
    return make_lists_column(input_column.size(),
                             std::move(offsets_column),
                             std::move(byte_column),
                             input_column.null_count(),
                             std::move(null_mask),
                             stream,
                             mr);
  }

  template <typename T, typename std::enable_if_t<std::is_integral<T>::value>* = nullptr>
  std::unique_ptr<column> operator()(column_view const& input_column,
                                     endianess_policy configuration,
                                     rmm::mr::device_memory_resource* mr,
                                     hipStream_t stream) const
  {
    size_type num_output_elements = input_column.size() * cudf::size_of(input_column.type());

    auto begin          = thrust::make_constant_iterator(cudf::size_of(input_column.type()));
    auto offsets_column = cudf::strings::detail::make_offsets_child_column(
      begin, begin + input_column.size(), mr, stream);

    auto byte_column = make_numeric_column(
      data_type{type_id::UINT8}, num_output_elements, mask_state::UNALLOCATED, stream, mr);
    auto bytes_view = byte_column->mutable_view();
    auto d_chars    = bytes_view.data<char>();
    auto d_data     = input_column.data<char>();

    rmm::device_buffer null_mask = copy_bitmask(input_column, stream, mr);

    if (configuration == endianess_policy::FLIP_ENDIANESS) {
      uint32_t mask = cudf::size_of(input_column.type()) - 1;
      thrust::for_each(rmm::exec_policy(stream)->on(stream),
                       thrust::make_counting_iterator(0),
                       thrust::make_counting_iterator(num_output_elements),
                       flip_endianness{d_chars, d_data, mask});
    } else {
      thrust::copy_n(rmm::exec_policy(stream)->on(stream), d_data, num_output_elements, d_chars);
    }

    return make_lists_column(input_column.size(),
                             std::move(offsets_column),
                             std::move(byte_column),
                             input_column.null_count(),
                             std::move(null_mask),
                             stream,
                             mr);
  }
};

template <>
std::unique_ptr<cudf::column> ByteListConversion::operator()<string_view>(
  column_view const& input_column,
  endianess_policy configuration,
  rmm::mr::device_memory_resource* mr,
  hipStream_t stream) const
{
  strings_column_view input_strings(input_column);
  auto strings_count = input_strings.size();
  if (strings_count == 0) return cudf::strings::detail::make_empty_strings_column(mr, stream);

  auto chars_column            = std::make_unique<column>(input_strings.chars(), stream, mr);
  auto offsets_column          = std::make_unique<column>(input_strings.offsets(), stream, mr);
  rmm::device_buffer null_mask = copy_bitmask(input_column, stream, mr);

  return make_lists_column(input_column.size(),
                           std::move(offsets_column),
                           std::move(chars_column),
                           input_column.null_count(),
                           std::move(null_mask),
                           stream,
                           mr);
}

}  // namespace detail

std::unique_ptr<column> byte_cast(column_view const& input_column,
                                  endianess_policy configuration,
                                  rmm::mr::device_memory_resource* mr,
                                  hipStream_t stream)
{
  CUDF_FUNC_RANGE();
  CUDF_EXPECTS(
    cudf::is_numeric(input_column.type()) || input_column.type().id() == cudf::type_id::STRING,
    "numeric type required to convert to big endian byte list");

  return type_dispatcher(
    input_column.type(), detail::ByteListConversion{}, input_column, configuration, mr, stream);
}

}  // namespace cudf
