/*
 * Copyright (c) 2020, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <cudf/copying.hpp>
#include <cudf/detail/gather.cuh>
#include <cudf/detail/nvtx/ranges.hpp>
#include <cudf/detail/reshape.hpp>
#include <cudf/table/table.hpp>
#include <cudf/types.hpp>
#include <cudf/utilities/error.hpp>

#include <rmm/cuda_stream_view.hpp>

#include <thrust/iterator/counting_iterator.h>
#include <thrust/iterator/transform_iterator.h>

#include <memory>

namespace cudf {
namespace {
struct tile_functor {
  size_type count;
  size_type __device__ operator()(size_type i) { return i % count; }
};

}  // anonymous namespace

namespace detail {
std::unique_ptr<table> tile(const table_view &in,
                            size_type count,
                            rmm::cuda_stream_view stream,
                            rmm::mr::device_memory_resource *mr)
{
  CUDF_EXPECTS(count >= 0, "Count cannot be negative");

  auto in_num_rows = in.num_rows();

  if (count == 0 or in_num_rows == 0) { return empty_like(in); }

  auto out_num_rows = in_num_rows * count;
  auto counting_it  = thrust::make_counting_iterator<size_type>(0);
  auto tiled_it     = thrust::make_transform_iterator(counting_it, tile_functor{in_num_rows});

  return detail::gather(in, tiled_it, tiled_it + out_num_rows, false, stream, mr);
}
}  // namespace detail

std::unique_ptr<table> tile(const table_view &in,
                            size_type count,
                            rmm::mr::device_memory_resource *mr)
{
  CUDF_FUNC_RANGE();
  return detail::tile(in, count, rmm::cuda_stream_default, mr);
}

}  // namespace cudf
