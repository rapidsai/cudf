/*
 * Copyright (c) 2025, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <cudf/detail/nvtx/ranges.hpp>
#include <cudf/detail/reshape.hpp>
#include <cudf/reshape.hpp>
#include <cudf/utilities/default_stream.hpp>
#include <cudf/utilities/error.hpp>
#include <cudf/utilities/type_dispatcher.hpp>

#include <rmm/cuda_stream_view.hpp>
#include <rmm/device_uvector.hpp>

#include <cub/device/device_memcpy.cuh>
#include <cuda/functional>
#include <hip/hip_runtime.h>
#include <thrust/device_vector.h>
#include <thrust/iterator/constant_iterator.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/iterator/transform_iterator.h>

namespace cudf {
namespace detail {
namespace {

template <typename T>
void _table_to_array(table_view const& input, void* output, rmm::cuda_stream_view stream)
{
  auto const num_columns = input.num_columns();
  auto const num_rows    = input.num_rows();
  auto const item_size   = sizeof(T);
  auto* base_ptr         = static_cast<cuda::std::byte*>(output);

  CUDF_EXPECTS(num_columns > 0, "Must have at least one column.");
  CUDF_EXPECTS(output != nullptr, "Output pointer cannot be null.");

  rmm::device_uvector<void*> d_srcs(num_columns, stream);
  rmm::device_uvector<void*> d_dsts(num_columns, stream);

  std::vector<void const*> h_srcs(num_columns);
  std::vector<void*> h_dsts(num_columns);

  for (int i = 0; i < num_columns; ++i) {
    auto const& col = input.column(i);
    CUDF_EXPECTS(col.type() == input.column(0).type(), "All columns must have the same dtype");
    CUDF_EXPECTS(col.null_count() == 0, "All columns must be non-nullable or contain no nulls");

    h_srcs[i] = static_cast<void const*>(col.data<T>());
    h_dsts[i] = static_cast<void*>(base_ptr + i * item_size * num_rows);
  }

  CUDF_CUDA_TRY(hipMemcpyAsync(d_srcs.data(),
                                h_srcs.data(),
                                sizeof(void*) * num_columns,
                                hipMemcpyHostToDevice,
                                stream.value()));
  CUDF_CUDA_TRY(hipMemcpyAsync(d_dsts.data(),
                                h_dsts.data(),
                                sizeof(void*) * num_columns,
                                hipMemcpyHostToDevice,
                                stream.value()));

  thrust::constant_iterator<size_t> sizes(static_cast<size_t>(item_size * num_rows));

  void* d_temp_storage      = nullptr;
  size_t temp_storage_bytes = 0;
  hipcub::DeviceMemcpy::Batched(d_temp_storage,
                             temp_storage_bytes,
                             d_srcs.begin(),
                             d_dsts.begin(),
                             sizes,
                             num_columns,
                             stream.value());

  rmm::device_buffer temp_storage(temp_storage_bytes, stream);

  hipcub::DeviceMemcpy::Batched(temp_storage.data(),
                             temp_storage_bytes,
                             d_srcs.begin(),
                             d_dsts.begin(),
                             sizes,
                             num_columns,
                             stream.value());
}

struct TableToArrayDispatcher {
  table_view const& input;
  void* output;
  rmm::cuda_stream_view stream;

  template <typename T, CUDF_ENABLE_IF(is_fixed_width<T>() || is_fixed_point<T>())>
  void operator()() const
  {
    if constexpr (is_fixed_point<T>()) {
      using StorageType = cudf::device_storage_type_t<T>;
      _table_to_array<StorageType>(input, output, stream);
    } else {
      _table_to_array<T>(input, output, stream);
    }
  }

  template <typename T, CUDF_ENABLE_IF(!is_fixed_width<T>() && !is_fixed_point<T>())>
  void operator()() const
  {
    CUDF_FAIL("Unsupported dtype");
  }
};

}  // namespace

void table_to_array(table_view const& input,
                    void* output,
                    data_type output_dtype,
                    rmm::cuda_stream_view stream)
{
  CUDF_EXPECTS(output != nullptr, "Output pointer cannot be null.");
  CUDF_EXPECTS(input.num_columns() > 0, "Input must have at least one column.");

  cudf::type_dispatcher(output_dtype, TableToArrayDispatcher{input, output, stream});
}

}  // namespace detail

void table_to_array(table_view const& input,
                    void* output,
                    data_type output_dtype,
                    rmm::cuda_stream_view stream)
{
  CUDF_FUNC_RANGE();
  cudf::detail::table_to_array(input, output, output_dtype, stream);
}

}  // namespace cudf
