/*
 * Copyright (c) 2020, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <cudf/copying.hpp>
#include <cudf/detail/gather.cuh>
#include <cudf/detail/nvtx/ranges.hpp>
#include <cudf/table/table_device_view.cuh>
#include <cudf/types.hpp>
#include <strings/utilities.cuh>

namespace cudf {
namespace detail {
namespace {
struct interleave_columns_functor {
  template <typename T, typename... Args>
  std::enable_if_t<not cudf::is_fixed_width<T>() and not std::is_same<T, cudf::string_view>::value,
                   std::unique_ptr<cudf::column>>
  operator()(Args&&... args)
  {
    CUDF_FAIL("interleave_columns not supported for dictionary and list types.");
  }

  template <typename T>
  std::enable_if_t<std::is_same<T, cudf::string_view>::value, std::unique_ptr<cudf::column>>
  operator()(table_view const& strings_columns,
             bool create_mask,
             rmm::mr::device_memory_resource* mr,
             hipStream_t stream = 0)
  {
    auto num_columns = strings_columns.num_columns();
    if (num_columns == 1)  // Single strings column returns a copy
      return std::make_unique<column>(*(strings_columns.begin()), stream, mr);

    auto strings_count = strings_columns.num_rows();
    if (strings_count == 0)  // All columns have 0 rows
      return strings::detail::make_empty_strings_column(mr, stream);

    // Create device views from the strings columns.
    auto table       = table_device_view::create(strings_columns, stream);
    auto d_table     = *table;
    auto num_strings = num_columns * strings_count;

    std::pair<rmm::device_buffer, size_type> valid_mask{{}, 0};
    if (create_mask) {
      // Create resulting null mask
      valid_mask = cudf::detail::valid_if(
        thrust::make_counting_iterator<size_type>(0),
        thrust::make_counting_iterator<size_type>(num_strings),
        [num_columns, d_table] __device__(size_type idx) {
          auto source_row_idx = idx % num_columns;
          auto source_col_idx = idx / num_columns;
          return !d_table.column(source_row_idx).is_null(source_col_idx);
        },
        stream,
        mr);
    }

    auto const null_count = valid_mask.second;

    // Build offsets column by computing sizes of each string in the output
    auto offsets_transformer = [num_columns, d_table] __device__(size_type idx) {
      // First compute the column and the row this item belongs to
      auto source_row_idx = idx % num_columns;
      auto source_col_idx = idx / num_columns;
      return d_table.column(source_row_idx).is_valid(source_col_idx)
               ? d_table.column(source_row_idx).element<string_view>(source_col_idx).size_bytes()
               : 0;
    };
    auto offsets_transformer_itr = thrust::make_transform_iterator(
      thrust::make_counting_iterator<size_type>(0), offsets_transformer);
    auto offsets_column = strings::detail::make_offsets_child_column(
      offsets_transformer_itr, offsets_transformer_itr + num_strings, mr, stream);
    auto d_results_offsets = offsets_column->view().template data<int32_t>();

    // Create the chars column
    size_type bytes = thrust::device_pointer_cast(d_results_offsets)[num_strings];
    auto chars_column =
      strings::detail::create_chars_child_column(num_strings, null_count, bytes, mr, stream);
    // Fill the chars column
    auto d_results_chars = chars_column->mutable_view().data<char>();
    thrust::for_each_n(
      rmm::exec_policy(stream)->on(stream),
      thrust::make_counting_iterator<size_type>(0),
      num_strings,
      [num_columns, d_table, d_results_offsets, d_results_chars] __device__(size_type idx) {
        auto source_row_idx = idx % num_columns;
        auto source_col_idx = idx / num_columns;

        // Do not write to buffer if the column value for this row is null
        if (d_table.column(source_row_idx).is_null(source_col_idx)) return;

        size_type offset = d_results_offsets[idx];
        char* d_buffer   = d_results_chars + offset;
        strings::detail::copy_string(
          d_buffer, d_table.column(source_row_idx).element<string_view>(source_col_idx));
      });

    return make_strings_column(num_strings,
                               std::move(offsets_column),
                               std::move(chars_column),
                               null_count,
                               std::move(valid_mask.first),
                               stream,
                               mr);
  }

  template <typename T>
  std::enable_if_t<cudf::is_fixed_width<T>(), std::unique_ptr<cudf::column>> operator()(
    table_view const& input,
    bool create_mask,
    rmm::mr::device_memory_resource* mr,
    hipStream_t stream = 0)
  {
    auto arch_column = input.column(0);
    auto output_size = input.num_columns() * input.num_rows();
    auto output =
      allocate_like(arch_column, output_size, mask_allocation_policy::NEVER, mr, stream);
    auto device_input  = table_device_view::create(input);
    auto device_output = mutable_column_device_view::create(*output);
    auto index_begin   = thrust::make_counting_iterator<size_type>(0);
    auto index_end     = thrust::make_counting_iterator<size_type>(output_size);

    using Type = device_storage_type_t<T>;

    auto func_value = [input   = *device_input,
                       divisor = input.num_columns()] __device__(size_type idx) {
      return input.column(idx % divisor).element<Type>(idx / divisor);
    };

    if (not create_mask) {
      thrust::transform(rmm::exec_policy(stream)->on(stream),
                        index_begin,
                        index_end,
                        device_output->begin<Type>(),
                        func_value);

      return output;
    }

    auto func_validity = [input   = *device_input,
                          divisor = input.num_columns()] __device__(size_type idx) {
      return input.column(idx % divisor).is_valid(idx / divisor);
    };

    thrust::transform_if(rmm::exec_policy(stream)->on(stream),
                         index_begin,
                         index_end,
                         device_output->begin<Type>(),
                         func_value,
                         func_validity);

    rmm::device_buffer mask;
    size_type null_count;

    std::tie(mask, null_count) = valid_if(index_begin, index_end, func_validity, stream, mr);

    output->set_null_mask(std::move(mask), null_count);

    return output;
  }
};

}  // anonymous namespace
}  // namespace detail

std::unique_ptr<column> interleave_columns(table_view const& input,
                                           rmm::mr::device_memory_resource* mr)
{
  CUDF_FUNC_RANGE();
  CUDF_EXPECTS(input.num_columns() > 0, "input must have at least one column to determine dtype.");

  auto const dtype = input.column(0).type();

  CUDF_EXPECTS(std::all_of(std::cbegin(input),
                           std::cend(input),
                           [dtype](auto const& col) { return dtype == col.type(); }),
               "DTYPE mismatch");

  auto const output_needs_mask = std::any_of(
    std::cbegin(input), std::cend(input), [](auto const& col) { return col.nullable(); });

  return type_dispatcher(dtype, detail::interleave_columns_functor{}, input, output_needs_mask, mr);
}

}  // namespace cudf
