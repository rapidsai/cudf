/*
 * Copyright (c) 2021-2022, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <cudf/column/column.hpp>
#include <cudf/detail/calendrical_month_sequence.cuh>
#include <cudf/detail/nvtx/ranges.hpp>
#include <cudf/scalar/scalar.hpp>
#include <cudf/utilities/default_stream.hpp>
#include <cudf/utilities/type_dispatcher.hpp>

#include <rmm/cuda_stream_view.hpp>
#include <rmm/exec_policy.hpp>

namespace cudf {
namespace detail {
std::unique_ptr<cudf::column> calendrical_month_sequence(size_type size,
                                                         scalar const& init,
                                                         size_type months,
                                                         rmm::cuda_stream_view stream,
                                                         rmm::mr::device_memory_resource* mr)
{
  return type_dispatcher(
    init.type(), calendrical_month_sequence_functor{}, size, init, months, stream, mr);
}
}  // namespace detail

std::unique_ptr<cudf::column> calendrical_month_sequence(size_type size,
                                                         scalar const& init,
                                                         size_type months,
                                                         rmm::mr::device_memory_resource* mr)
{
  CUDF_FUNC_RANGE();
  return detail::calendrical_month_sequence(size, init, months, cudf::get_default_stream(), mr);
}

}  // namespace cudf
