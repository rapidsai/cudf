#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019-2020, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <cudf/column/column.hpp>
#include <cudf/column/column_factories.hpp>
#include <cudf/column/column_view.hpp>
#include <cudf/copying.hpp>
#include <cudf/detail/gather.cuh>
#include <cudf/detail/nvtx/ranges.hpp>
#include <cudf/detail/repeat.hpp>
#include <cudf/filling.hpp>
#include <cudf/scalar/scalar.hpp>
#include <cudf/table/table.hpp>
#include <cudf/table/table_view.hpp>
#include <cudf/types.hpp>
#include <cudf/utilities/type_dispatcher.hpp>

#include <rmm/cuda_stream_view.hpp>
#include <rmm/device_vector.hpp>
#include <rmm/exec_policy.hpp>
#include <rmm/mr/device/per_device_resource.hpp>

#include <thrust/binary_search.h>
#include <thrust/iterator/constant_iterator.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/iterator/transform_output_iterator.h>
#include <thrust/scan.h>

#include <limits>
#include <memory>

namespace {
struct count_accessor {
  cudf::scalar const* p_scalar = nullptr;

  template <typename T>
  std::enable_if_t<std::is_integral<T>::value, cudf::size_type> operator()(
    rmm::cuda_stream_view stream)
  {
    using ScalarType = cudf::scalar_type_t<T>;
#if 1
    // TODO: temporary till cudf::scalar's value() function is marked as const
    auto p_count = const_cast<ScalarType*>(static_cast<ScalarType const*>(this->p_scalar));
#else
    auto p_count = static_cast<ScalarType const*>(this->p_scalar);
#endif
    auto count = p_count->value();
    // static_cast is necessary due to bool
    CUDF_EXPECTS(static_cast<int64_t>(count) <= std::numeric_limits<cudf::size_type>::max(),
                 "count should not exceed size_type's limit.");
    return static_cast<cudf::size_type>(count);
  }

  template <typename T>
  std::enable_if_t<not std::is_integral<T>::value, cudf::size_type> operator()(
    rmm::cuda_stream_view)
  {
    CUDF_FAIL("count value should be a integral type.");
  }
};

struct compute_offsets {
  cudf::column_view const* p_column = nullptr;

  template <typename T>
  std::enable_if_t<std::is_integral<T>::value, rmm::device_vector<cudf::size_type>> operator()(
    bool check_count, rmm::cuda_stream_view stream)
  {
    // static_cast is necessary due to bool
    if (check_count && static_cast<int64_t>(std::numeric_limits<T>::max()) >
                         std::numeric_limits<cudf::size_type>::max()) {
      auto max = thrust::reduce(p_column->begin<T>(), p_column->end<T>(), 0, thrust::maximum<T>());
      CUDF_EXPECTS(max <= std::numeric_limits<cudf::size_type>::max(),
                   "count should not have values larger than size_type's limit.");
    }
    rmm::device_vector<cudf::size_type> offsets(p_column->size());
    thrust::inclusive_scan(
      rmm::exec_policy(stream), p_column->begin<T>(), p_column->end<T>(), offsets.begin());
    if (check_count == true) {
      CUDF_EXPECTS(
        thrust::is_sorted(rmm::exec_policy(stream), offsets.begin(), offsets.end()) == true,
        "count has negative values or the resulting table has more \
                    rows than size_type's limit.");
    }

    return offsets;
  }

  template <typename T>
  std::enable_if_t<not std::is_integral<T>::value, rmm::device_vector<cudf::size_type>> operator()(
    bool check_count, rmm::cuda_stream_view stream)
  {
    CUDF_FAIL("count value should be a integral type.");
  }
};

}  // namespace

namespace cudf {
namespace detail {
std::unique_ptr<table> repeat(table_view const& input_table,
                              column_view const& count,
                              bool check_count,
                              rmm::cuda_stream_view stream,
                              rmm::mr::device_memory_resource* mr)
{
  CUDF_EXPECTS(input_table.num_rows() == count.size(), "in and count must have equal size");
  CUDF_EXPECTS(count.has_nulls() == false, "count cannot contain nulls");

  if (input_table.num_rows() == 0) { return cudf::empty_like(input_table); }

  auto offsets = cudf::type_dispatcher(count.type(), compute_offsets{&count}, check_count, stream);

  size_type output_size{offsets.back()};
  rmm::device_vector<size_type> indices(output_size);
  thrust::upper_bound(rmm::exec_policy(stream),
                      offsets.begin(),
                      offsets.end(),
                      thrust::make_counting_iterator(0),
                      thrust::make_counting_iterator(output_size),
                      indices.begin());

  return gather(
    input_table, indices.begin(), indices.end(), out_of_bounds_policy::DONT_CHECK, stream, mr);
}

std::unique_ptr<table> repeat(table_view const& input_table,
                              size_type count,
                              rmm::cuda_stream_view stream,
                              rmm::mr::device_memory_resource* mr)
{
  CUDF_EXPECTS(count >= 0, "count value should be non-negative");
  CUDF_EXPECTS(
    static_cast<int64_t>(input_table.num_rows()) * count <= std::numeric_limits<size_type>::max(),
    "The resulting table has more rows than size_type's limit.");

  if ((input_table.num_rows() == 0) || (count == 0)) { return cudf::empty_like(input_table); }

  auto output_size = input_table.num_rows() * count;
  auto map_begin   = thrust::make_transform_iterator(
    thrust::make_counting_iterator(0), [count] __device__(auto i) { return i / count; });
  auto map_end = map_begin + output_size;

  return gather(input_table, map_begin, map_end, out_of_bounds_policy::DONT_CHECK, stream, mr);
}

}  // namespace detail

std::unique_ptr<table> repeat(table_view const& input_table,
                              column_view const& count,
                              bool check_count,
                              rmm::mr::device_memory_resource* mr)
{
  CUDF_FUNC_RANGE();
  return detail::repeat(input_table, count, check_count, rmm::cuda_stream_default, mr);
}

std::unique_ptr<table> repeat(table_view const& input_table,
                              size_type count,
                              rmm::mr::device_memory_resource* mr)
{
  CUDF_FUNC_RANGE();
  return detail::repeat(input_table, count, rmm::cuda_stream_default, mr);
}

}  // namespace cudf
