/*
 * Copyright (c) 2019, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <cudf/utilities/error.hpp>
#include <utilities/column_utils.hpp>
#include <cudf/utilities/legacy/type_dispatcher.hpp>
#include <cudf/legacy/copying.hpp>

#include <cudf/cudf.h>
#include <cudf/types.h>
#include <rmm/thrust_rmm_allocator.h>

#include <thrust/iterator/counting_iterator.h>
#include <thrust/iterator/constant_iterator.h>
#include <thrust/iterator/transform_output_iterator.h>
#include <thrust/scan.h>
#include <thrust/binary_search.h>


namespace cudf {

namespace detail {

cudf::table repeat(const cudf::table &in, const gdf_column& count, hipStream_t stream = 0) {
  CUDF_EXPECTS(count.dtype == gdf_dtype_of<cudf::size_type>(),
    "Count column should be of index type");
  CUDF_EXPECTS(in.num_rows() == count.size, "in and count must have equal size");
  CUDF_EXPECTS(not has_nulls(count), "count cannot contain nulls");

  if (in.num_rows() == 0) {
    return cudf::empty_like(in);
  }
  
  auto exec_policy = rmm::exec_policy(stream)->on(stream);
  rmm::device_vector<cudf::size_type> offset(count.size);
  auto count_data = static_cast <cudf::size_type*> (count.data);
  
  thrust::inclusive_scan(exec_policy, count_data, count_data + count.size, offset.begin());

  cudf::size_type output_size = offset.back();

  rmm::device_vector<cudf::size_type> indices(output_size);
  thrust::upper_bound(exec_policy,
                      offset.begin(), offset.end(),
                      thrust::make_counting_iterator(0),
                      thrust::make_counting_iterator(output_size),
                      indices.begin());

  cudf::table output = cudf::allocate_like(in, output_size, RETAIN, stream);

  cudf::gather(&in, indices.data().get(), &output);

  return output;
}

cudf::table repeat(const cudf::table &in, const gdf_scalar& count, hipStream_t stream = 0) {
  CUDF_EXPECTS(count.dtype == gdf_dtype_of<cudf::size_type>(),
    "Count value should be of index type");
  CUDF_EXPECTS(count.is_valid, "count cannot be null");

  if (in.num_rows() == 0) {
    return cudf::empty_like(in);
  }
  
  cudf::size_type stride = count.data.si32;

  cudf::size_type output_size = stride * in.num_rows();
  auto offset = thrust::make_transform_iterator(
    thrust::make_counting_iterator(0),
    [stride] __device__ (auto i) { return (i+1) * stride; }
  );

  rmm::device_vector<cudf::size_type> indices(output_size);
  thrust::upper_bound(rmm::exec_policy(stream)->on(stream),
                      offset, offset + in.num_rows(),
                      thrust::make_counting_iterator(0),
                      thrust::make_counting_iterator(output_size),
                      indices.begin());

  cudf::table output = cudf::allocate_like(in, output_size, RETAIN, stream);

  cudf::gather(&in, indices.data().get(), &output);

  return output;
}

} // namespace detail


cudf::table repeat(const cudf::table &in, const gdf_column& count) {
  return detail::repeat(in, count);
}

cudf::table repeat(const cudf::table &in, const gdf_scalar& count) {
  return detail::repeat(in, count);
}

} // namespace cudf
