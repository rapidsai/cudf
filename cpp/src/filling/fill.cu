/*
 * Copyright (c) 2019-2020, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <cudf/column/column.hpp>
#include <cudf/column/column_factories.hpp>
#include <cudf/detail/copy_range.cuh>
#include <cudf/detail/fill.hpp>
#include <cudf/detail/nvtx/ranges.hpp>
#include <cudf/dictionary/detail/encode.hpp>
#include <cudf/dictionary/detail/search.hpp>
#include <cudf/dictionary/detail/update_keys.hpp>
#include <cudf/dictionary/dictionary_column_view.hpp>
#include <cudf/dictionary/dictionary_factories.hpp>
#include <cudf/filling.hpp>
#include <cudf/scalar/scalar.hpp>
#include <cudf/strings/detail/fill.hpp>
#include <cudf/types.hpp>
#include <cudf/utilities/error.hpp>
#include <cudf/utilities/traits.hpp>

#include <hip/hip_runtime.h>

#include <memory>

namespace {
template <typename T>
void in_place_fill(cudf::mutable_column_view& destination,
                   cudf::size_type begin,
                   cudf::size_type end,
                   cudf::scalar const& value,
                   hipStream_t stream = 0)
{
  using ScalarType = cudf::scalar_type_t<T>;
  auto p_scalar    = static_cast<ScalarType const*>(&value);
  T fill_value     = p_scalar->value(stream);
  bool is_valid    = p_scalar->is_valid();
  cudf::detail::copy_range(thrust::make_constant_iterator(fill_value),
                           thrust::make_constant_iterator(is_valid),
                           destination,
                           begin,
                           end,
                           stream);
}

struct in_place_fill_range_dispatch {
  cudf::scalar const& value;
  cudf::mutable_column_view& destination;

  template <typename T>
  std::enable_if_t<cudf::is_fixed_width<T>(), void> operator()(cudf::size_type begin,
                                                               cudf::size_type end,
                                                               hipStream_t stream = 0)
  {
    in_place_fill<T>(destination, begin, end, value, stream);
  }

  template <typename T>
  std::enable_if_t<not cudf::is_fixed_width<T>(), void> operator()(cudf::size_type begin,
                                                                   cudf::size_type end,
                                                                   hipStream_t stream = 0)
  {
    CUDF_FAIL("in-place fill does not work for variable width types.");
  }
};

struct out_of_place_fill_range_dispatch {
  cudf::scalar const& value;
  cudf::column_view const& input;

  template <typename T>
  std::unique_ptr<cudf::column> operator()(
    cudf::size_type begin,
    cudf::size_type end,
    rmm::mr::device_memory_resource* mr = rmm::mr::get_default_resource(),
    hipStream_t stream                 = 0)
  {
    CUDF_EXPECTS(input.type() == value.type(), "Data type mismatch.");
    auto p_ret = std::make_unique<cudf::column>(input, stream, mr);

    if (end != begin) {  // otherwise no fill
      if (!p_ret->nullable() && !value.is_valid()) {
        p_ret->set_null_mask(
          cudf::create_null_mask(p_ret->size(), cudf::mask_state::ALL_VALID, stream, mr), 0);
      }

      auto ret_view = p_ret->mutable_view();
      in_place_fill<T>(ret_view, begin, end, value, stream);
    }

    return p_ret;
  }
};

template <>
std::unique_ptr<cudf::column> out_of_place_fill_range_dispatch::operator()<cudf::list_view>(
  cudf::size_type begin,
  cudf::size_type end,
  rmm::mr::device_memory_resource* mr,
  hipStream_t stream)
{
  CUDF_FAIL("list_view not supported yet");
}

template <>
std::unique_ptr<cudf::column> out_of_place_fill_range_dispatch::operator()<cudf::string_view>(
  cudf::size_type begin,
  cudf::size_type end,
  rmm::mr::device_memory_resource* mr,
  hipStream_t stream)
{
  CUDF_EXPECTS(input.type() == value.type(), "Data type mismatch.");
  using ScalarType = cudf::scalar_type_t<cudf::string_view>;
  auto p_scalar    = static_cast<ScalarType const*>(&value);
  return cudf::strings::detail::fill(
    cudf::strings_column_view(input), begin, end, *p_scalar, mr, stream);
}

template <>
std::unique_ptr<cudf::column> out_of_place_fill_range_dispatch::operator()<cudf::dictionary32>(
  cudf::size_type begin,
  cudf::size_type end,
  rmm::mr::device_memory_resource* mr,
  hipStream_t stream)
{
  if (input.size() == 0) return std::make_unique<cudf::column>(input, stream, mr);
  cudf::dictionary_column_view const target(input);
  CUDF_EXPECTS(target.keys().type() == value.type(), "Data type mismatch.");

  // if the scalar is invalid, then just copy the column and fill the null mask
  if (!value.is_valid()) {
    auto result = std::make_unique<cudf::column>(input, stream, mr);
    auto mview  = result->mutable_view();
    cudf::set_null_mask(mview.null_mask(), begin, end, false, stream);
    mview.set_null_count(input.null_count() + (end - begin));
    return result;
  }

  // add the scalar to get the output dictionary key-set
  auto scalar_column =
    cudf::make_column_from_scalar(value, 1, rmm::mr::get_default_resource(), stream);
  auto target_matched =
    cudf::dictionary::detail::add_keys(target, scalar_column->view(), mr, stream);
  cudf::column_view const target_indices =
    cudf::dictionary_column_view(target_matched->view()).get_indices_annotated();

  // get the index of the key just added
  auto index_of_value = cudf::dictionary::detail::get_index(
    target_matched->view(), value, rmm::mr::get_default_resource(), stream);
  // now call fill using just the indices column and the new index
  out_of_place_fill_range_dispatch filler{*index_of_value, target_indices};
  auto new_indices       = filler.template operator()<int32_t>(begin, end, mr, stream);
  auto const output_size = new_indices->size();        // record these
  auto const null_count  = new_indices->null_count();  // before the release()
  auto contents          = new_indices->release();
  // create the new indices column from the result
  auto indices_column = std::make_unique<cudf::column>(cudf::data_type{cudf::type_id::INT32},
                                                       static_cast<cudf::size_type>(output_size),
                                                       std::move(*(contents.data.release())),
                                                       rmm::device_buffer{0, stream, mr},
                                                       0);

  // take the keys from matched column
  std::unique_ptr<cudf::column> keys_column(std::move(target_matched->release().children.back()));

  // create column with keys_column and indices_column
  return cudf::make_dictionary_column(std::move(keys_column),
                                      std::move(indices_column),
                                      std::move(*(contents.null_mask.release())),
                                      null_count);
}

}  // namespace

namespace cudf {
namespace detail {
void fill_in_place(mutable_column_view& destination,
                   size_type begin,
                   size_type end,
                   scalar const& value,
                   hipStream_t stream)
{
  CUDF_EXPECTS(cudf::is_fixed_width(destination.type()) == true,
               "In-place fill does not support variable-sized types.");
  CUDF_EXPECTS((begin >= 0) && (end <= destination.size()) && (begin <= end),
               "Range is out of bounds.");
  CUDF_EXPECTS((destination.nullable() == true) || (value.is_valid() == true),
               "destination should be nullable or value should be non-null.");
  CUDF_EXPECTS(destination.type() == value.type(), "Data type mismatch.");

  if (end != begin) {  // otherwise no-op
    cudf::type_dispatcher(
      destination.type(), in_place_fill_range_dispatch{value, destination}, begin, end, stream);
  }

  return;
}

std::unique_ptr<column> fill(column_view const& input,
                             size_type begin,
                             size_type end,
                             scalar const& value,
                             rmm::mr::device_memory_resource* mr,
                             hipStream_t stream)
{
  CUDF_EXPECTS((begin >= 0) && (end <= input.size()) && (begin <= end), "Range is out of bounds.");

  return cudf::type_dispatcher(
    input.type(), out_of_place_fill_range_dispatch{value, input}, begin, end, mr, stream);
}

}  // namespace detail

void fill_in_place(mutable_column_view& destination,
                   size_type begin,
                   size_type end,
                   scalar const& value)
{
  CUDF_FUNC_RANGE();
  return detail::fill_in_place(destination, begin, end, value, 0);
}

std::unique_ptr<column> fill(column_view const& input,
                             size_type begin,
                             size_type end,
                             scalar const& value,
                             rmm::mr::device_memory_resource* mr)
{
  CUDF_FUNC_RANGE();
  return detail::fill(input, begin, end, value, mr, 0);
}

}  // namespace cudf
