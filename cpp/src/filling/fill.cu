/*
 * Copyright (c) 2019, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <cudf/filling.hpp>
#include <cudf/types.hpp>
#include <cudf/column/column.hpp>
#include <cudf/column/column_view.hpp>
#include <cudf/column/column_factories.hpp>
#include <cudf/detail/copy_range.cuh>
#include <cudf/detail/fill.hpp>
#include <cudf/scalar/scalar.hpp>
#include <cudf/strings/detail/fill.hpp>
#include <cudf/utilities/error.hpp>
#include <cudf/utilities/traits.hpp>

#include <hip/hip_runtime.h>

#include <memory>

namespace {

template <typename T>
void in_place_fill(cudf::mutable_column_view& destination,
                   cudf::size_type begin,
                   cudf::size_type end,
                   cudf::scalar const& value,
                   hipStream_t stream = 0) {
    using ScalarType = cudf::experimental::scalar_type_t<T>;
    auto p_scalar = static_cast<ScalarType const*>(&value);
    T fill_value = p_scalar->value(stream);
    bool is_valid = p_scalar->is_valid();
    cudf::experimental::detail::copy_range(
      thrust::make_constant_iterator(fill_value),
      thrust::make_constant_iterator(is_valid),
      destination, begin, end, stream);
}

struct in_place_fill_range_dispatch {
  cudf::scalar const& value;
  cudf::mutable_column_view& destination;

  template <typename T>
  std::enable_if_t<cudf::is_fixed_width<T>(), void>
  operator()(cudf::size_type begin, cudf::size_type end,
             hipStream_t stream = 0) {
    in_place_fill<T>(destination, begin, end, value, stream);
  }

  template <typename T>
  std::enable_if_t<not cudf::is_fixed_width<T>(), void>
  operator()(cudf::size_type begin, cudf::size_type end,
             hipStream_t stream = 0) {
    CUDF_FAIL("in-place fill does not work for variable width types.");
  }
};

struct out_of_place_fill_range_dispatch {
  cudf::scalar const& value;
  cudf::column_view const& input;

  template <typename T>
  std::enable_if_t<cudf::is_fixed_width<T>(), std::unique_ptr<cudf::column>>
  operator()(
      cudf::size_type begin, cudf::size_type end,
      rmm::mr::device_memory_resource* mr = rmm::mr::get_default_resource(), 
      hipStream_t stream = 0) {
    auto p_ret = std::make_unique<cudf::column>(input, stream, mr);

    if (end != begin) {  // otherwise no fill
      if (!p_ret->nullable() && !value.is_valid()) {
        p_ret->set_null_mask(
          cudf::create_null_mask(p_ret->size(), cudf::mask_state::ALL_VALID, stream, mr), 0);
      }

      auto ret_view = p_ret->mutable_view();
      in_place_fill<T>(ret_view, begin, end, value, stream);
    }

    return p_ret;
  }

  template <typename T>
  std::enable_if_t<std::is_same<cudf::string_view, T>::value,
                   std::unique_ptr<cudf::column>>
  operator()(
      cudf::size_type begin, cudf::size_type end,
      rmm::mr::device_memory_resource* mr = rmm::mr::get_default_resource(), 
      hipStream_t stream = 0) {
    using ScalarType = cudf::experimental::scalar_type_t<T>;
    auto p_scalar = static_cast<ScalarType const*>(&value);
    return cudf::strings::detail::fill(cudf::strings_column_view(input),
                                       begin, end, *p_scalar, mr, stream);
  }
  
  template <typename T>
  std::enable_if_t<std::is_same<cudf::dictionary32, T>::value,
                   std::unique_ptr<cudf::column>>
  operator()(
      cudf::size_type begin, cudf::size_type end,
      rmm::mr::device_memory_resource* mr = rmm::mr::get_default_resource(), 
      hipStream_t stream = 0) {
        CUDF_FAIL("dictionary not supported yet");
  }
};

}  // namespace

namespace cudf {
namespace experimental {

namespace detail {

void fill_in_place(mutable_column_view& destination,
                   size_type begin,
                   size_type end,
                   scalar const& value,
                   hipStream_t stream) {
  CUDF_EXPECTS(cudf::is_fixed_width(destination.type()) == true,
               "In-place fill does not support variable-sized types.");
  CUDF_EXPECTS((begin >= 0) &&
               (begin <= end) &&
               (begin < destination.size()) &&
               (end <= destination.size()),
               "Range is out of bounds.");
  CUDF_EXPECTS((destination.nullable() == true) || (value.is_valid() == true),
               "destination should be nullable or value should be non-null.");
  CUDF_EXPECTS(destination.type() == value.type(), "Data type mismatch.");

  if (end != begin) {  // otherwise no-op
    cudf::experimental::type_dispatcher(
      destination.type(),
      in_place_fill_range_dispatch{value, destination},
      begin, end, stream);
  }

  return;
}

std::unique_ptr<column> fill(column_view const& input,
                             size_type begin,
                             size_type end,
                             scalar const& value,
                             rmm::mr::device_memory_resource* mr,
                             hipStream_t stream) {
  CUDF_EXPECTS((begin >= 0) &&
               (begin <= end) &&
               (begin < input.size()) &&
               (end <= input.size()),
               "Range is out of bounds.");
  CUDF_EXPECTS(input.type() == value.type(), "Data type mismatch.");

  return cudf::experimental::type_dispatcher(
      input.type(),
      out_of_place_fill_range_dispatch{value, input},
      begin, end, mr, stream);
}

}  // namespace detail

void fill_in_place(mutable_column_view& destination,
                   size_type begin,
                   size_type end,
                   scalar const& value) {
  return detail::fill_in_place(destination, begin, end, value, 0);
}

std::unique_ptr<column> fill(column_view const& input,
                             size_type begin,
                             size_type end,
                             scalar const& value,
                             rmm::mr::device_memory_resource* mr) {
  return detail::fill(input, begin, end, value, mr, 0);
}

}  // namespace experimental
}  // namespace cudf
