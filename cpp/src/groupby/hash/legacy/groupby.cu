#include "hip/hip_runtime.h"

/*
 * Copyright (c) 2019, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <cudf/cudf.h>
#include <rmm/thrust_rmm_allocator.h>
#include <thrust/fill.h>
#include <bitmask/legacy/bit_mask.cuh>
#include <cudf/detail/utilities/integer_utils.hpp>
#include <cudf/legacy/bitmask.hpp>
#include <cudf/legacy/copying.hpp>
#include <cudf/legacy/groupby.hpp>
#include <cudf/legacy/table.hpp>
#include <cudf/utilities/legacy/nvcategory_util.hpp>
#include <cudf/utilities/legacy/type_dispatcher.hpp>
#include <hash/concurrent_unordered_map.cuh>
#include <table/legacy/device_table.cuh>
#include <table/legacy/device_table_row_operators.cuh>
#include <type_traits>
#include <utilities/legacy/column_utils.hpp>
#include <utilities/legacy/cuda_utils.hpp>
#include <vector>
#include "groupby/common/legacy/aggregation_requests.hpp"
#include "groupby/common/legacy/type_info.hpp"
#include "groupby/common/legacy/utils.hpp"
#include "groupby_kernels.cuh"

namespace cudf {
namespace groupby {

namespace hash {
namespace {

template <bool keys_have_nulls, bool values_have_nulls>
auto build_aggregation_map(table const& input_keys,
                           table const& input_values,
                           device_table const& d_input_keys,
                           device_table const& d_input_values,
                           std::vector<operators> const& ops,
                           Options options,
                           hipStream_t stream) {
  cudf::size_type constexpr unused_key{std::numeric_limits<cudf::size_type>::max()};
  cudf::size_type constexpr unused_value{std::numeric_limits<cudf::size_type>::max()};
  CUDF_EXPECTS(input_keys.num_rows() < unused_key, "Groupby input size too large.");

  // The exact output size is unknown a priori, therefore, use the input size as
  // an upper bound.
  cudf::size_type const output_size_estimate{input_keys.num_rows()};

  cudf::table sparse_output_values{output_size_estimate,
                                   target_dtypes(column_dtypes(input_values), ops),
                                   column_dtype_infos(input_values),
                                   values_have_nulls,
                                   false,
                                   stream};

  initialize_with_identity(sparse_output_values, ops, stream);

  auto d_sparse_output_values = device_table::create(sparse_output_values, stream);
  rmm::device_vector<operators> d_ops(ops);

  // If we ignore null keys, then nulls are not equivalent
  bool const null_keys_are_equal{not options.ignore_null_keys};
  bool const skip_key_rows_with_nulls{keys_have_nulls and not null_keys_are_equal};

  row_hasher<keys_have_nulls> hasher{d_input_keys};
  row_equality_comparator<keys_have_nulls> rows_equal{
    d_input_keys, d_input_keys, null_keys_are_equal};

  using map_type       = concurrent_unordered_map<cudf::size_type,
                                            cudf::size_type,
                                            decltype(hasher),
                                            decltype(rows_equal)>;
  using allocator_type = typename map_type::allocator_type;

  auto map = map_type::create(compute_hash_table_size(input_keys.num_rows()),
                              unused_key,
                              unused_value,
                              hasher,
                              rows_equal,
                              allocator_type(),
                              stream);

  // TODO: Explore optimal block size and work per thread.
  cudf::util::cuda::grid_config_1d grid_params{input_keys.num_rows(), 256};

  if (skip_key_rows_with_nulls) {
    auto row_bitmask{cudf::row_bitmask(input_keys, stream)};
    build_aggregation_map<true, values_have_nulls>
      <<<grid_params.num_blocks, grid_params.num_threads_per_block, 0, stream>>>(
        *map,
        d_input_keys,
        d_input_values,
        *d_sparse_output_values,
        d_ops.data().get(),
        row_bitmask.data().get());
  } else {
    build_aggregation_map<false, values_have_nulls>
      <<<grid_params.num_blocks, grid_params.num_threads_per_block, 0, stream>>>(
        *map, d_input_keys, d_input_values, *d_sparse_output_values, d_ops.data().get(), nullptr);
  }
  CHECK_CUDA(stream);

  return std::make_pair(std::move(map), std::move(sparse_output_values));
}

template <bool keys_have_nulls, bool values_have_nulls, typename Map>
auto extract_results(table const& input_keys,
                     table const& input_values,
                     device_table const& d_input_keys,
                     table const& sparse_output_values,
                     Map const& map,
                     hipStream_t stream) {
  cudf::table output_keys{
    cudf::allocate_like(input_keys, keys_have_nulls ? RETAIN : NEVER, stream)};
  cudf::table output_values{
    cudf::allocate_like(sparse_output_values, values_have_nulls ? RETAIN : NEVER, stream)};

  auto d_sparse_output_values = device_table::create(sparse_output_values, stream);

  auto d_output_keys   = device_table::create(output_keys, stream);
  auto d_output_values = device_table::create(output_values, stream);

  cudf::size_type* d_result_size{nullptr};
  RMM_TRY(RMM_ALLOC(&d_result_size, sizeof(cudf::size_type), stream));
  CUDA_TRY(hipMemsetAsync(d_result_size, 0, sizeof(cudf::size_type), stream));

  cudf::util::cuda::grid_config_1d grid_params{input_keys.num_rows(), 256};

  extract_groupby_result<keys_have_nulls, values_have_nulls>
    <<<grid_params.num_blocks, grid_params.num_threads_per_block, 0, stream>>>(
      map, d_input_keys, *d_output_keys, *d_sparse_output_values, *d_output_values, d_result_size);

  CHECK_CUDA(stream);

  cudf::size_type result_size{-1};
  CUDA_TRY(hipMemcpyAsync(
    &result_size, d_result_size, sizeof(cudf::size_type), hipMemcpyDeviceToHost, stream));

  // Update size and null count of output columns
  auto update_column = [result_size](gdf_column* col) {
    CUDF_EXPECTS(col != nullptr, "Attempt to update Null column.");
    col->size = result_size;
    set_null_count(*col);
    return col;
  };

  std::transform(output_keys.begin(), output_keys.end(), output_keys.begin(), update_column);
  std::transform(output_values.begin(), output_values.end(), output_values.begin(), update_column);

  return std::make_pair(std::move(output_keys), std::move(output_values));
}

/**---------------------------------------------------------------------------*
 * @brief Computes the groupby operation for a set of keys, values, and
 * operators using a hash-based implementation.
 *
 * The algorithm has two primary steps:
 * 1.) Build a hash map
 * 2.) Extract the non-empty entries from the hash table
 *
 * 1.) The hash map is built by inserting every row `i` from the `keys` and
 * `values` tables as a single (key,value) pair. When the pair is inserted, if
 * the key was not already present in the map, then the corresponding value is
 * simply copied to the output. If the key was already present in the map,
 * then the inserted `values` row is aggregated with the existing row. This
 * aggregation is done for every element `j` in the row by applying aggregation
 * operation `j` between the new and existing element.
 *
 * This process yields a hash map and table holding the resulting aggregation
 * rows. The aggregation output table is sparse, i.e., not every row is
 * populated. This is because the size of the output is not known a priori, and
 * so the output aggregation table is allocated to be as large as the input (the
 * upper bound of the output size).
 *
 * 2.) The final result is materialized by extracting the non-empty keys from
 * the hash map and the non-empty rows from the sparse output aggregation table.
 * Every non-empty key and value row is appended to the output key and value
 * tables.
 *
 * @tparam keys_have_nulls Indicates keys have one or more null values
 * @tparam values_have_nulls Indicates values have one or more null values
 * @param keys Table whose rows are used as keys of the groupby
 * @param values Table whose rows are aggregated in the groupby
 * @param ops Set of aggregation operations to perform for each element in a row
 * in the values table
 * @param options Options to control behavior of the groupby operation
 * @param stream CUDA stream on which all memory allocations and kernels will be
 * executed
 * @return A pair of the output keys table and output values table
 *---------------------------------------------------------------------------**/
template <bool keys_have_nulls, bool values_have_nulls>
auto compute_hash_groupby(cudf::table const& keys,
                          cudf::table const& values,
                          std::vector<operators> const& ops,
                          Options options,
                          hipStream_t stream) {
  CUDF_EXPECTS(values.num_columns() == static_cast<cudf::size_type>(ops.size()),
               "Size mismatch between number of value columns and number of "
               "aggregations.");

  // An "aggregation request" is the combination of a `gdf_column*` to a column
  // of values, and an aggregation operation enum indicating the aggregation
  // requested to be performed on the column
  std::vector<AggRequestType> original_requests(values.num_columns());
  std::transform(values.begin(),
                 values.end(),
                 ops.begin(),
                 original_requests.begin(),
                 [](gdf_column const* col, operators op) {
                   return std::make_pair(const_cast<gdf_column*>(col), op);
                 });

  // Some aggregations are "compound", meaning they need be satisfied via the
  // composition of 1 or more "simple" aggregation requests. For example, MEAN
  // is satisfied via the division of the SUM by the COUNT aggregation. We
  // translate these compound requests into simple requests, and compute the
  // groupby operation for these simple requests. Later, we translate the simple
  // requests back to compound request results.
  std::vector<SimpleAggRequestCounter> simple_agg_columns = compound_to_simple(original_requests);

  std::vector<gdf_column*> simple_values_columns;
  std::vector<operators> simple_operators;
  for (auto const& p : simple_agg_columns) {
    const AggRequestType& agg_req_type = p.first;
    simple_values_columns.push_back(const_cast<gdf_column*>(agg_req_type.first));
    simple_operators.push_back(agg_req_type.second);
  }

  cudf::table simple_values_table{simple_values_columns};

  auto const d_input_keys   = device_table::create(keys);
  auto const d_input_values = device_table::create(simple_values_table);

  // Step 1: Build hash map
  auto result = build_aggregation_map<keys_have_nulls, values_have_nulls>(
    keys, simple_values_table, *d_input_keys, *d_input_values, simple_operators, options, stream);

  auto const map{std::move(result.first)};
  cudf::table sparse_output_values{result.second};

  // Step 2: Extract non-empty entries
  cudf::table output_keys;
  cudf::table simple_output_values;
  std::tie(output_keys, simple_output_values) = extract_results<keys_have_nulls, values_have_nulls>(
    keys, values, *d_input_keys, sparse_output_values, *map, stream);

  // Delete intermediate results storage
  sparse_output_values.destroy();

  // If any of the original requests were compound, compute them from the
  // results of simple aggregation requests
  cudf::table final_output_values =
    compute_original_requests(original_requests, simple_agg_columns, simple_output_values, stream);

  return std::make_pair(std::move(output_keys), std::move(final_output_values));
}

/**---------------------------------------------------------------------------*
 * @brief Returns appropriate callable instantiation of `compute_hash_groupby`
 * based on presence of null values in keys and values.
 *
 * @param keys The groupby key columns
 * @param values The groupby value columns
 * @return Instantiated callable of compute_hash_groupby
 *---------------------------------------------------------------------------**/
auto groupby_null_specialization(table const& keys, table const& values) {
  if (cudf::has_nulls(keys)) {
    if (cudf::has_nulls(values)) {
      return compute_hash_groupby<true, true>;
    } else {
      return compute_hash_groupby<true, false>;
    }
  } else {
    if (cudf::has_nulls(values)) {
      return compute_hash_groupby<false, true>;
    } else {
      return compute_hash_groupby<false, false>;
    }
  }
}

}  // namespace
namespace detail {
std::pair<cudf::table, cudf::table> groupby(cudf::table const& keys,
                                            cudf::table const& values,
                                            std::vector<operators> const& ops,
                                            Options options,
                                            hipStream_t stream = 0) {
  CUDF_EXPECTS(keys.num_rows() == values.num_rows(),
               "Size mismatch between number of rows in keys and values.");

  verify_operators(values, ops);

  // Empty inputs
  if (keys.num_rows() == 0) {
    return std::make_pair(
      cudf::empty_like(keys),
      cudf::table(0, target_dtypes(column_dtypes(values), ops), column_dtype_infos(values)));
  }

  auto compute_groupby = groupby_null_specialization(keys, values);

  cudf::table output_keys;
  cudf::table output_values;
  std::tie(output_keys, output_values) = compute_groupby(keys, values, ops, options, stream);

  update_nvcategories(keys, output_keys, values, output_values);

  return std::make_pair(std::move(output_keys), std::move(output_values));
}
}  // namespace detail

std::pair<cudf::table, cudf::table> groupby(cudf::table const& keys,
                                            cudf::table const& values,
                                            std::vector<operators> const& ops,
                                            Options options) {
  return detail::groupby(keys, values, ops, options);
}
}  // namespace hash
}  // namespace groupby
}  // namespace cudf
