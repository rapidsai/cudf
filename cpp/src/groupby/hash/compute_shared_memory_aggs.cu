#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2024, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "compute_shared_memory_aggs.hpp"
#include "global_memory_aggregator.cuh"
#include "helpers.cuh"
#include "shared_memory_aggregator.cuh"
#include "single_pass_functors.cuh"

#include <cudf/aggregation.hpp>
#include <cudf/detail/utilities/cuda.cuh>
#include <cudf/detail/utilities/cuda.hpp>
#include <cudf/detail/utilities/integer_utils.hpp>
#include <cudf/table/table_device_view.cuh>
#include <cudf/types.hpp>
#include <cudf/utilities/bit.hpp>

#include <rmm/cuda_stream_view.hpp>

#include <hip/hip_cooperative_groups.h>
#include <cuda/std/cstddef>

namespace cudf::groupby::detail::hash {
namespace {
/// Functor used by type dispatcher returning the size of the underlying C++ type
struct size_of_functor {
  template <typename T>
  __device__ constexpr cudf::size_type operator()()
  {
    return sizeof(T);
  }
};

// Prepares shared memory data required by each output column, exits if
// no enough memory space to perform the shared memory aggregation for the
// current output column
__device__ void calculate_columns_to_aggregate(cudf::size_type& col_start,
                                               cudf::size_type& col_end,
                                               cudf::mutable_table_device_view output_values,
                                               cudf::size_type output_size,
                                               cudf::size_type* shmem_agg_res_offsets,
                                               cudf::size_type* shmem_agg_mask_offsets,
                                               cudf::size_type cardinality,
                                               cudf::size_type total_agg_size)
{
  col_start                       = col_end;
  cudf::size_type bytes_allocated = 0;

  auto const valid_col_size = round_to_multiple_of_8(sizeof(bool) * cardinality);

  while (bytes_allocated < total_agg_size && col_end < output_size) {
    auto const col_idx = col_end;
    auto const next_col_size =
      round_to_multiple_of_8(cudf::type_dispatcher<cudf::dispatch_storage_type>(
                               output_values.column(col_idx).type(), size_of_functor{}) *
                             cardinality);
    auto const next_col_total_size = next_col_size + valid_col_size;

    if (bytes_allocated + next_col_total_size > total_agg_size) {
      CUDF_UNREACHABLE("No enough memory space for shared memory aggregations");
    }

    shmem_agg_res_offsets[col_end]  = bytes_allocated;
    shmem_agg_mask_offsets[col_end] = bytes_allocated + next_col_size;

    bytes_allocated += next_col_total_size;
    ++col_end;
  }
}

// Each block initialize its own shared memory aggregation results
__device__ void initialize_shmem_aggregations(cooperative_groups::thread_block const& block,
                                              cudf::size_type col_start,
                                              cudf::size_type col_end,
                                              cudf::mutable_table_device_view output_values,
                                              cuda::std::byte* shmem_agg_storage,
                                              cudf::size_type* shmem_agg_res_offsets,
                                              cudf::size_type* shmem_agg_mask_offsets,
                                              cudf::size_type cardinality,
                                              cudf::aggregation::Kind const* d_agg_kinds)
{
  for (auto col_idx = col_start; col_idx < col_end; col_idx++) {
    for (auto idx = block.thread_rank(); idx < cardinality; idx += block.num_threads()) {
      cuda::std::byte* target =
        reinterpret_cast<cuda::std::byte*>(shmem_agg_storage + shmem_agg_res_offsets[col_idx]);
      bool* target_mask =
        reinterpret_cast<bool*>(shmem_agg_storage + shmem_agg_mask_offsets[col_idx]);
      cudf::detail::dispatch_type_and_aggregation(output_values.column(col_idx).type(),
                                                  d_agg_kinds[col_idx],
                                                  initialize_shmem{},
                                                  target,
                                                  target_mask,
                                                  idx);
    }
  }
  block.sync();
}

__device__ void compute_pre_aggregrations(cudf::size_type col_start,
                                          cudf::size_type col_end,
                                          bitmask_type const* row_bitmask,
                                          bool skip_rows_with_nulls,
                                          cudf::table_device_view source,
                                          cudf::size_type num_input_rows,
                                          cudf::size_type* local_mapping_index,
                                          cuda::std::byte* shmem_agg_storage,
                                          cudf::size_type* shmem_agg_res_offsets,
                                          cudf::size_type* shmem_agg_mask_offsets,
                                          cudf::aggregation::Kind const* d_agg_kinds)
{
  // Aggregates global memory sources to shared memory targets
  for (auto source_idx = cudf::detail::grid_1d::global_thread_id(); source_idx < num_input_rows;
       source_idx += cudf::detail::grid_1d::grid_stride()) {
    if (not skip_rows_with_nulls or cudf::bit_is_set(row_bitmask, source_idx)) {
      auto const target_idx = local_mapping_index[source_idx];
      for (auto col_idx = col_start; col_idx < col_end; col_idx++) {
        auto const source_col = source.column(col_idx);

        cuda::std::byte* target =
          reinterpret_cast<cuda::std::byte*>(shmem_agg_storage + shmem_agg_res_offsets[col_idx]);
        bool* target_mask =
          reinterpret_cast<bool*>(shmem_agg_storage + shmem_agg_mask_offsets[col_idx]);

        cudf::detail::dispatch_type_and_aggregation(source_col.type(),
                                                    d_agg_kinds[col_idx],
                                                    shmem_element_aggregator{},
                                                    target,
                                                    target_mask,
                                                    target_idx,
                                                    source_col,
                                                    source_idx);
      }
    }
  }
}

__device__ void compute_final_aggregations(cooperative_groups::thread_block const& block,
                                           cudf::size_type col_start,
                                           cudf::size_type col_end,
                                           cudf::table_device_view input_values,
                                           cudf::mutable_table_device_view target,
                                           cudf::size_type cardinality,
                                           cudf::size_type* global_mapping_index,
                                           cuda::std::byte* shmem_agg_storage,
                                           cudf::size_type* agg_res_offsets,
                                           cudf::size_type* agg_mask_offsets,
                                           cudf::aggregation::Kind const* d_agg_kinds)
{
  // Aggregates shared memory sources to global memory targets
  for (auto idx = block.thread_rank(); idx < cardinality; idx += block.num_threads()) {
    auto const target_idx =
      global_mapping_index[block.group_index().x * GROUPBY_SHM_MAX_ELEMENTS + idx];
    for (auto col_idx = col_start; col_idx < col_end; col_idx++) {
      auto target_col = target.column(col_idx);

      cuda::std::byte* source =
        reinterpret_cast<cuda::std::byte*>(shmem_agg_storage + agg_res_offsets[col_idx]);
      bool* source_mask = reinterpret_cast<bool*>(shmem_agg_storage + agg_mask_offsets[col_idx]);

      cudf::detail::dispatch_type_and_aggregation(input_values.column(col_idx).type(),
                                                  d_agg_kinds[col_idx],
                                                  gmem_element_aggregator{},
                                                  target_col,
                                                  target_idx,
                                                  input_values.column(col_idx),
                                                  source,
                                                  source_mask,
                                                  idx);
    }
  }
  block.sync();
}

/* Takes the local_mapping_index and global_mapping_index to compute
 * pre (shared) and final (global) aggregates*/
CUDF_KERNEL void single_pass_shmem_aggs_kernel(cudf::size_type num_rows,
                                               bitmask_type const* row_bitmask,
                                               bool skip_rows_with_nulls,
                                               cudf::size_type* local_mapping_index,
                                               cudf::size_type* global_mapping_index,
                                               cudf::size_type* block_cardinality,
                                               cudf::table_device_view input_values,
                                               cudf::mutable_table_device_view output_values,
                                               cudf::aggregation::Kind const* d_agg_kinds,
                                               cudf::size_type total_agg_size,
                                               cudf::size_type offsets_size)
{
  auto const block       = cooperative_groups::this_thread_block();
  auto const cardinality = block_cardinality[block.group_index().x];
  if (cardinality >= GROUPBY_CARDINALITY_THRESHOLD) { return; }

  auto const num_cols = output_values.num_columns();

  __shared__ cudf::size_type col_start;
  __shared__ cudf::size_type col_end;
  extern __shared__ cuda::std::byte shmem_agg_storage[];

  cudf::size_type* shmem_agg_res_offsets =
    reinterpret_cast<cudf::size_type*>(shmem_agg_storage + total_agg_size);
  cudf::size_type* shmem_agg_mask_offsets =
    reinterpret_cast<cudf::size_type*>(shmem_agg_storage + total_agg_size + offsets_size);

  if (block.thread_rank() == 0) {
    col_start = 0;
    col_end   = 0;
  }
  block.sync();

  while (col_end < num_cols) {
    if (block.thread_rank() == 0) {
      calculate_columns_to_aggregate(col_start,
                                     col_end,
                                     output_values,
                                     num_cols,
                                     shmem_agg_res_offsets,
                                     shmem_agg_mask_offsets,
                                     cardinality,
                                     total_agg_size);
    }
    block.sync();

    initialize_shmem_aggregations(block,
                                  col_start,
                                  col_end,
                                  output_values,
                                  shmem_agg_storage,
                                  shmem_agg_res_offsets,
                                  shmem_agg_mask_offsets,
                                  cardinality,
                                  d_agg_kinds);

    compute_pre_aggregrations(col_start,
                              col_end,
                              row_bitmask,
                              skip_rows_with_nulls,
                              input_values,
                              num_rows,
                              local_mapping_index,
                              shmem_agg_storage,
                              shmem_agg_res_offsets,
                              shmem_agg_mask_offsets,
                              d_agg_kinds);
    block.sync();

    compute_final_aggregations(block,
                               col_start,
                               col_end,
                               input_values,
                               output_values,
                               cardinality,
                               global_mapping_index,
                               shmem_agg_storage,
                               shmem_agg_res_offsets,
                               shmem_agg_mask_offsets,
                               d_agg_kinds);
  }
}

constexpr size_t get_previous_multiple_of_8(size_t number) { return number / 8 * 8; }

}  // namespace

size_t available_shared_memory_size(cudf::size_type grid_size)
{
  auto const active_blocks_per_sm =
    cudf::util::div_rounding_up_safe(grid_size, cudf::detail::num_multiprocessors());

  size_t dynamic_shmem_size = 0;
  CUDF_CUDA_TRY(cudaOccupancyAvailableDynamicSMemPerBlock(
    &dynamic_shmem_size, single_pass_shmem_aggs_kernel, active_blocks_per_sm, GROUPBY_BLOCK_SIZE));
  return get_previous_multiple_of_8(0.5 * dynamic_shmem_size);
}

size_t shmem_offsets_size(cudf::size_type num_cols) { return sizeof(cudf::size_type) * num_cols; }

void compute_shared_memory_aggs(cudf::size_type grid_size,
                                cudf::size_type num_input_rows,
                                bitmask_type const* row_bitmask,
                                bool skip_rows_with_nulls,
                                cudf::size_type* local_mapping_index,
                                cudf::size_type* global_mapping_index,
                                cudf::size_type* block_cardinality,
                                cudf::table_device_view input_values,
                                cudf::mutable_table_device_view output_values,
                                cudf::aggregation::Kind const* d_agg_kinds,
                                rmm::cuda_stream_view stream)
{
  auto const shmem_size = available_shared_memory_size(grid_size);
  // For each aggregation, need one offset determining where the aggregation is
  // performed, another indicating the validity of the aggregation
  auto const offsets_size = shmem_offsets_size(output_values.num_columns());
  // The rest of shmem is utilized for the actual arrays in shmem
  CUDF_EXPECTS(shmem_size > offsets_size * 2, "No enough space for shared memory aggregations");
  auto const shmem_agg_size = shmem_size - offsets_size * 2;
  single_pass_shmem_aggs_kernel<<<grid_size, GROUPBY_BLOCK_SIZE, shmem_size, stream>>>(
    num_input_rows,
    row_bitmask,
    skip_rows_with_nulls,
    local_mapping_index,
    global_mapping_index,
    block_cardinality,
    input_values,
    output_values,
    d_agg_kinds,
    shmem_agg_size,
    offsets_size);
}
}  // namespace cudf::groupby::detail::hash
