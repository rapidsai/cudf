#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019-2022, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <groupby/common/utils.hpp>
#include <groupby/hash/groupby_kernels.cuh>

#include <cudf/aggregation.hpp>
#include <cudf/column/column.hpp>
#include <cudf/column/column_factories.hpp>
#include <cudf/column/column_view.hpp>
#include <cudf/copying.hpp>
#include <cudf/detail/aggregation/aggregation.cuh>
#include <cudf/detail/aggregation/aggregation.hpp>
#include <cudf/detail/aggregation/result_cache.hpp>
#include <cudf/detail/binaryop.hpp>
#include <cudf/detail/gather.hpp>
#include <cudf/detail/groupby.hpp>
#include <cudf/detail/null_mask.hpp>
#include <cudf/detail/replace.hpp>
#include <cudf/detail/unary.hpp>
#include <cudf/detail/utilities/cuda.cuh>
#include <cudf/detail/utilities/hash_functions.cuh>
#include <cudf/detail/utilities/vector_factories.hpp>
#include <cudf/dictionary/dictionary_column_view.hpp>
#include <cudf/groupby.hpp>
#include <cudf/scalar/scalar.hpp>
#include <cudf/table/experimental/row_operators.cuh>
#include <cudf/table/table.hpp>
#include <cudf/table/table_device_view.cuh>
#include <cudf/table/table_view.hpp>
#include <cudf/types.hpp>
#include <cudf/utilities/traits.cuh>
#include <cudf/utilities/traits.hpp>
#include <hash/concurrent_unordered_map.cuh>

#include <rmm/cuda_stream_view.hpp>

#include <thrust/copy.h>
#include <thrust/for_each.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/iterator/transform_iterator.h>

#include <memory>
#include <unordered_set>
#include <utility>

#include <cuda/std/atomic>

namespace cudf {
namespace groupby {
namespace detail {
namespace hash {
namespace {

// TODO: replace it with `cuco::static_map`
// https://github.com/rapidsai/cudf/issues/10401
using map_type = concurrent_unordered_map<
  cudf::size_type,
  cudf::size_type,
  cudf::experimental::row::hash::device_row_hasher<cudf::detail::default_hash,
                                                   cudf::nullate::DYNAMIC>,
  cudf::experimental::row::equality::device_row_comparator<cudf::nullate::DYNAMIC>>;

/**
 * @brief List of aggregation operations that can be computed with a hash-based
 * implementation.
 */
constexpr std::array<aggregation::Kind, 12> hash_aggregations{aggregation::SUM,
                                                              aggregation::PRODUCT,
                                                              aggregation::MIN,
                                                              aggregation::MAX,
                                                              aggregation::COUNT_VALID,
                                                              aggregation::COUNT_ALL,
                                                              aggregation::ARGMIN,
                                                              aggregation::ARGMAX,
                                                              aggregation::SUM_OF_SQUARES,
                                                              aggregation::MEAN,
                                                              aggregation::STD,
                                                              aggregation::VARIANCE};

// Could be hash: SUM, PRODUCT, MIN, MAX, COUNT_VALID, COUNT_ALL, ANY, ALL,
// Compound: MEAN(SUM, COUNT_VALID), VARIANCE, STD(MEAN (SUM, COUNT_VALID), COUNT_VALID),
// ARGMAX, ARGMIN

// TODO replace with std::find in C++20 onwards.
template <class T, size_t N>
constexpr bool array_contains(std::array<T, N> const& haystack, T needle)
{
  for (auto const& val : haystack) {
    if (val == needle) return true;
  }
  return false;
}

/**
 * @brief Indicates whether the specified aggregation operation can be computed
 * with a hash-based implementation.
 *
 * @param t The aggregation operation to verify
 * @return true `t` is valid for a hash based groupby
 * @return false `t` is invalid for a hash based groupby
 */
bool constexpr is_hash_aggregation(aggregation::Kind t)
{
  return array_contains(hash_aggregations, t);
}

class groupby_simple_aggregations_collector final
  : public cudf::detail::simple_aggregations_collector {
 public:
  using cudf::detail::simple_aggregations_collector::visit;

  std::vector<std::unique_ptr<aggregation>> visit(data_type col_type,
                                                  cudf::detail::min_aggregation const&) override
  {
    std::vector<std::unique_ptr<aggregation>> aggs;
    aggs.push_back(col_type.id() == type_id::STRING ? make_argmin_aggregation()
                                                    : make_min_aggregation());
    return aggs;
  }

  std::vector<std::unique_ptr<aggregation>> visit(data_type col_type,
                                                  cudf::detail::max_aggregation const&) override
  {
    std::vector<std::unique_ptr<aggregation>> aggs;
    aggs.push_back(col_type.id() == type_id::STRING ? make_argmax_aggregation()
                                                    : make_max_aggregation());
    return aggs;
  }

  std::vector<std::unique_ptr<aggregation>> visit(data_type col_type,
                                                  cudf::detail::mean_aggregation const&) override
  {
    (void)col_type;
    CUDF_EXPECTS(is_fixed_width(col_type), "MEAN aggregation expects fixed width type");
    std::vector<std::unique_ptr<aggregation>> aggs;
    aggs.push_back(make_sum_aggregation());
    // COUNT_VALID
    aggs.push_back(make_count_aggregation());

    return aggs;
  }

  std::vector<std::unique_ptr<aggregation>> visit(data_type,
                                                  cudf::detail::var_aggregation const&) override
  {
    std::vector<std::unique_ptr<aggregation>> aggs;
    aggs.push_back(make_sum_aggregation());
    // COUNT_VALID
    aggs.push_back(make_count_aggregation());

    return aggs;
  }

  std::vector<std::unique_ptr<aggregation>> visit(data_type,
                                                  cudf::detail::std_aggregation const&) override
  {
    std::vector<std::unique_ptr<aggregation>> aggs;
    aggs.push_back(make_sum_aggregation());
    // COUNT_VALID
    aggs.push_back(make_count_aggregation());

    return aggs;
  }

  std::vector<std::unique_ptr<aggregation>> visit(
    data_type, cudf::detail::correlation_aggregation const&) override
  {
    std::vector<std::unique_ptr<aggregation>> aggs;
    aggs.push_back(make_sum_aggregation());
    // COUNT_VALID
    aggs.push_back(make_count_aggregation());

    return aggs;
  }
};

class hash_compound_agg_finalizer final : public cudf::detail::aggregation_finalizer {
  column_view col;
  data_type result_type;
  cudf::detail::result_cache* sparse_results;
  cudf::detail::result_cache* dense_results;
  device_span<size_type const> gather_map;
  map_type const& map;
  bitmask_type const* __restrict__ row_bitmask;
  rmm::cuda_stream_view stream;
  rmm::mr::device_memory_resource* mr;

 public:
  using cudf::detail::aggregation_finalizer::visit;

  hash_compound_agg_finalizer(column_view col,
                              cudf::detail::result_cache* sparse_results,
                              cudf::detail::result_cache* dense_results,
                              device_span<size_type const> gather_map,
                              map_type const& map,
                              bitmask_type const* row_bitmask,
                              rmm::cuda_stream_view stream,
                              rmm::mr::device_memory_resource* mr)
    : col(col),
      sparse_results(sparse_results),
      dense_results(dense_results),
      gather_map(gather_map),
      map(map),
      row_bitmask(row_bitmask),
      stream(stream),
      mr(mr)
  {
    result_type = cudf::is_dictionary(col.type()) ? cudf::dictionary_column_view(col).keys().type()
                                                  : col.type();
  }

  auto to_dense_agg_result(cudf::aggregation const& agg)
  {
    auto s                  = sparse_results->get_result(col, agg);
    auto dense_result_table = cudf::detail::gather(table_view({std::move(s)}),
                                                   gather_map,
                                                   out_of_bounds_policy::DONT_CHECK,
                                                   cudf::detail::negative_index_policy::NOT_ALLOWED,
                                                   stream,
                                                   mr);
    return std::move(dense_result_table->release()[0]);
  }

  // Enables conversion of ARGMIN/ARGMAX into MIN/MAX
  auto gather_argminmax(aggregation const& agg)
  {
    auto arg_result = to_dense_agg_result(agg);
    // We make a view of ARG(MIN/MAX) result without a null mask and gather
    // using this map. The values in data buffer of ARG(MIN/MAX) result
    // corresponding to null values was initialized to ARG(MIN/MAX)_SENTINEL
    // which is an out of bounds index value (-1) and causes the gathered
    // value to be null.
    column_view null_removed_map(
      data_type(type_to_id<size_type>()),
      arg_result->size(),
      static_cast<void const*>(arg_result->view().template data<size_type>()));
    auto gather_argminmax =
      cudf::detail::gather(table_view({col}),
                           null_removed_map,
                           arg_result->nullable() ? cudf::out_of_bounds_policy::NULLIFY
                                                  : cudf::out_of_bounds_policy::DONT_CHECK,
                           cudf::detail::negative_index_policy::NOT_ALLOWED,
                           stream,
                           mr);
    return std::move(gather_argminmax->release()[0]);
  }

  // Declare overloads for each kind of aggregation to dispatch
  void visit(cudf::aggregation const& agg) override
  {
    if (dense_results->has_result(col, agg)) return;
    dense_results->add_result(col, agg, to_dense_agg_result(agg));
  }

  void visit(cudf::detail::min_aggregation const& agg) override
  {
    if (dense_results->has_result(col, agg)) return;
    if (result_type.id() == type_id::STRING) {
      auto transformed_agg = make_argmin_aggregation();
      dense_results->add_result(col, agg, gather_argminmax(*transformed_agg));
    } else {
      dense_results->add_result(col, agg, to_dense_agg_result(agg));
    }
  }

  void visit(cudf::detail::max_aggregation const& agg) override
  {
    if (dense_results->has_result(col, agg)) return;

    if (result_type.id() == type_id::STRING) {
      auto transformed_agg = make_argmax_aggregation();
      dense_results->add_result(col, agg, gather_argminmax(*transformed_agg));
    } else {
      dense_results->add_result(col, agg, to_dense_agg_result(agg));
    }
  }

  void visit(cudf::detail::mean_aggregation const& agg) override
  {
    if (dense_results->has_result(col, agg)) return;

    auto sum_agg   = make_sum_aggregation();
    auto count_agg = make_count_aggregation();
    this->visit(*sum_agg);
    this->visit(*count_agg);
    column_view sum_result   = dense_results->get_result(col, *sum_agg);
    column_view count_result = dense_results->get_result(col, *count_agg);

    auto result =
      cudf::detail::binary_operation(sum_result,
                                     count_result,
                                     binary_operator::DIV,
                                     cudf::detail::target_type(result_type, aggregation::MEAN),
                                     stream,
                                     mr);
    dense_results->add_result(col, agg, std::move(result));
  }

  void visit(cudf::detail::var_aggregation const& agg) override
  {
    if (dense_results->has_result(col, agg)) return;

    auto sum_agg   = make_sum_aggregation();
    auto count_agg = make_count_aggregation();
    this->visit(*sum_agg);
    this->visit(*count_agg);
    column_view sum_result   = sparse_results->get_result(col, *sum_agg);
    column_view count_result = sparse_results->get_result(col, *count_agg);

    auto values_view = column_device_view::create(col, stream);
    auto sum_view    = column_device_view::create(sum_result, stream);
    auto count_view  = column_device_view::create(count_result, stream);

    auto var_result = make_fixed_width_column(
      cudf::detail::target_type(result_type, agg.kind), col.size(), mask_state::ALL_NULL, stream);
    auto var_result_view = mutable_column_device_view::create(var_result->mutable_view(), stream);
    mutable_table_view var_table_view{{var_result->mutable_view()}};
    cudf::detail::initialize_with_identity(var_table_view, {agg.kind}, stream);

    thrust::for_each_n(
      rmm::exec_policy(stream),
      thrust::make_counting_iterator(0),
      col.size(),
      ::cudf::detail::var_hash_functor<map_type>{
        map, row_bitmask, *var_result_view, *values_view, *sum_view, *count_view, agg._ddof});
    sparse_results->add_result(col, agg, std::move(var_result));
    dense_results->add_result(col, agg, to_dense_agg_result(agg));
  }

  void visit(cudf::detail::std_aggregation const& agg) override
  {
    if (dense_results->has_result(col, agg)) return;
    auto var_agg = make_variance_aggregation(agg._ddof);
    this->visit(*dynamic_cast<cudf::detail::var_aggregation*>(var_agg.get()));
    column_view variance = dense_results->get_result(col, *var_agg);

    auto result = cudf::detail::unary_operation(variance, unary_operator::SQRT, stream, mr);
    dense_results->add_result(col, agg, std::move(result));
  }
};
// flatten aggs to filter in single pass aggs
std::tuple<table_view, std::vector<aggregation::Kind>, std::vector<std::unique_ptr<aggregation>>>
flatten_single_pass_aggs(host_span<aggregation_request const> requests)
{
  std::vector<column_view> columns;
  std::vector<std::unique_ptr<aggregation>> aggs;
  std::vector<aggregation::Kind> agg_kinds;

  for (auto const& request : requests) {
    auto const& agg_v = request.aggregations;

    std::unordered_set<aggregation::Kind> agg_kinds_set;
    auto insert_agg = [&](column_view const& request_values, std::unique_ptr<aggregation>&& agg) {
      if (agg_kinds_set.insert(agg->kind).second) {
        agg_kinds.push_back(agg->kind);
        aggs.push_back(std::move(agg));
        columns.push_back(request_values);
      }
    };

    auto values_type = cudf::is_dictionary(request.values.type())
                         ? cudf::dictionary_column_view(request.values).keys().type()
                         : request.values.type();
    for (auto&& agg : agg_v) {
      groupby_simple_aggregations_collector collector;

      for (auto& agg_s : agg->get_simple_aggregations(values_type, collector)) {
        insert_agg(request.values, std::move(agg_s));
      }
    }
  }

  return std::make_tuple(table_view(columns), std::move(agg_kinds), std::move(aggs));
}

/**
 * @brief Gather sparse results into dense using `gather_map` and add to
 * `dense_cache`
 *
 * @see groupby_null_templated()
 */
void sparse_to_dense_results(table_view const& keys,
                             host_span<aggregation_request const> requests,
                             cudf::detail::result_cache* sparse_results,
                             cudf::detail::result_cache* dense_results,
                             device_span<size_type const> gather_map,
                             map_type const& map,
                             bool keys_have_nulls,
                             null_policy include_null_keys,
                             rmm::cuda_stream_view stream,
                             rmm::mr::device_memory_resource* mr)
{
  auto row_bitmask = bitmask_and(keys, stream, rmm::mr::get_current_device_resource()).first;
  bool skip_key_rows_with_nulls = keys_have_nulls and include_null_keys == null_policy::EXCLUDE;
  bitmask_type const* row_bitmask_ptr =
    skip_key_rows_with_nulls ? static_cast<bitmask_type*>(row_bitmask.data()) : nullptr;

  for (auto const& request : requests) {
    auto const& agg_v = request.aggregations;
    auto const& col   = request.values;

    // Given an aggregation, this will get the result from sparse_results and
    // convert and return dense, compacted result
    auto finalizer = hash_compound_agg_finalizer(
      col, sparse_results, dense_results, gather_map, map, row_bitmask_ptr, stream, mr);
    for (auto&& agg : agg_v) {
      agg->finalize(finalizer);
    }
  }
}

// make table that will hold sparse results
auto create_sparse_results_table(table_view const& flattened_values,
                                 std::vector<aggregation::Kind> aggs,
                                 rmm::cuda_stream_view stream)
{
  // TODO single allocation - room for performance improvement
  std::vector<std::unique_ptr<column>> sparse_columns;
  std::transform(
    flattened_values.begin(),
    flattened_values.end(),
    aggs.begin(),
    std::back_inserter(sparse_columns),
    [stream](auto const& col, auto const& agg) {
      bool nullable =
        (agg == aggregation::COUNT_VALID or agg == aggregation::COUNT_ALL)
          ? false
          : (col.has_nulls() or agg == aggregation::VARIANCE or agg == aggregation::STD);
      auto mask_flag = (nullable) ? mask_state::ALL_NULL : mask_state::UNALLOCATED;

      auto col_type = cudf::is_dictionary(col.type())
                        ? cudf::dictionary_column_view(col).keys().type()
                        : col.type();

      return make_fixed_width_column(
        cudf::detail::target_type(col_type, agg), col.size(), mask_flag, stream);
    });

  table sparse_table(std::move(sparse_columns));
  mutable_table_view table_view = sparse_table.mutable_view();
  cudf::detail::initialize_with_identity(table_view, aggs, stream);
  return sparse_table;
}

/**
 * @brief Computes all aggregations from `requests` that require a single pass
 * over the data and stores the results in `sparse_results`
 */
void compute_single_pass_aggs(table_view const& keys,
                              host_span<aggregation_request const> requests,
                              cudf::detail::result_cache* sparse_results,
                              map_type& map,
                              bool keys_have_nulls,
                              null_policy include_null_keys,
                              rmm::cuda_stream_view stream)
{
  // flatten the aggs to a table that can be operated on by aggregate_row
  auto const [flattened_values, agg_kinds, aggs] = flatten_single_pass_aggs(requests);

  // make table that will hold sparse results
  table sparse_table = create_sparse_results_table(flattened_values, agg_kinds, stream);
  // prepare to launch kernel to do the actual aggregation
  auto d_sparse_table = mutable_table_device_view::create(sparse_table, stream);
  auto d_values       = table_device_view::create(flattened_values, stream);
  auto const d_aggs   = cudf::detail::make_device_uvector_async(agg_kinds, stream);
  auto const skip_key_rows_with_nulls =
    keys_have_nulls and include_null_keys == null_policy::EXCLUDE;

  auto row_bitmask =
    skip_key_rows_with_nulls ? cudf::detail::bitmask_and(keys, stream).first : rmm::device_buffer{};

  thrust::for_each_n(
    rmm::exec_policy(stream),
    thrust::make_counting_iterator(0),
    keys.num_rows(),
    hash::compute_single_pass_aggs_fn<map_type>{map,
                                                *d_values,
                                                *d_sparse_table,
                                                d_aggs.data(),
                                                static_cast<bitmask_type*>(row_bitmask.data()),
                                                skip_key_rows_with_nulls});
  // Add results back to sparse_results cache
  auto sparse_result_cols = sparse_table.release();
  for (size_t i = 0; i < aggs.size(); i++) {
    // Note that the cache will make a copy of this temporary aggregation
    sparse_results->add_result(
      flattened_values.column(i), *aggs[i], std::move(sparse_result_cols[i]));
  }
}

/**
 * @brief Computes and returns a device vector containing all populated keys in
 * `map`.
 */
rmm::device_uvector<size_type> extract_populated_keys(map_type const& map,
                                                      size_type num_keys,
                                                      rmm::cuda_stream_view stream)
{
  rmm::device_uvector<size_type> populated_keys(num_keys, stream);

  auto const get_key = [] __device__(auto const& element) { return element.first; };  // first = key
  auto const key_used = [unused = map.get_unused_key()] __device__(auto key) {
    return key != unused;
  };
  auto key_itr = thrust::make_transform_iterator(map.data(), get_key);

  // thrust::copy_if has a bug where it cannot iterate over int-max values
  // so if map.capacity() > int-max we'll call thrust::copy_if in chunks instead
  auto const copy_size =
    std::min(map.capacity(), static_cast<std::size_t>(std::numeric_limits<int>::max()));
  auto const key_end = key_itr + map.capacity();
  auto pop_keys_itr  = populated_keys.begin();

  std::size_t output_size = 0;
  while (key_itr != key_end) {
    auto const copy_end = static_cast<std::size_t>(std::distance(key_itr, key_end)) <= copy_size
                            ? key_end
                            : key_itr + copy_size;
    auto const end_it =
      thrust::copy_if(rmm::exec_policy(stream), key_itr, copy_end, pop_keys_itr, key_used);
    auto const copied = std::distance(pop_keys_itr, end_it);
    pop_keys_itr += copied;
    output_size += copied;
    key_itr = copy_end;
  }

  populated_keys.resize(output_size, stream);
  return populated_keys;
}

/**
 * @brief Computes groupby using hash table.
 *
 * First, we create a hash table that stores the indices of unique rows in
 * `keys`. The upper limit on the number of values in this map is the number
 * of rows in `keys`.
 *
 * To store the results of aggregations, we create temporary sparse columns
 * which have the same size as input value columns. Using the hash map, we
 * determine the location within the sparse column to write the result of the
 * aggregation into.
 *
 * The sparse column results of all aggregations are stored into the cache
 * `sparse_results`. This enables the use of previously calculated results in
 * other aggregations.
 *
 * All the aggregations which can be computed in a single pass are computed
 * first, in a combined kernel. Then using these results, aggregations that
 * require multiple passes, will be computed.
 *
 * Finally, using the hash map, we generate a vector of indices of populated
 * values in sparse result columns. Then, for each aggregation originally
 * requested in `requests`, we gather sparse results into a column of dense
 * results using the aforementioned index vector. Dense results are stored into
 * the in/out parameter `cache`.
 */
std::unique_ptr<table> groupby(table_view const& keys,
                               host_span<aggregation_request const> requests,
                               cudf::detail::result_cache* cache,
                               bool const keys_have_nulls,
                               null_policy const include_null_keys,
                               rmm::cuda_stream_view stream,
                               rmm::mr::device_memory_resource* mr)
{
  auto const num_keys            = keys.num_rows();
  auto const null_keys_are_equal = null_equality::EQUAL;
  auto const has_null            = nullate::DYNAMIC{cudf::has_nested_nulls(keys)};

  auto preprocessed_keys = cudf::experimental::row::hash::preprocessed_table::create(keys, stream);
  auto const comparator  = cudf::experimental::row::equality::self_comparator{preprocessed_keys};
  auto const row_hash    = cudf::experimental::row::hash::row_hasher{std::move(preprocessed_keys)};
  auto const d_key_equal = comparator.equal_to(has_null, null_keys_are_equal);
  auto const d_row_hash  = row_hash.device_hasher(has_null);

  size_type constexpr unused_key{std::numeric_limits<size_type>::max()};
  size_type constexpr unused_value{std::numeric_limits<size_type>::max()};

  using allocator_type = typename map_type::allocator_type;

  auto map = map_type::create(compute_hash_table_size(num_keys),
                              stream,
                              unused_key,
                              unused_value,
                              d_row_hash,
                              d_key_equal,
                              allocator_type());

  // Cache of sparse results where the location of aggregate value in each
  // column is indexed by the hash map
  cudf::detail::result_cache sparse_results(requests.size());

  // Compute all single pass aggs first
  compute_single_pass_aggs(
    keys, requests, &sparse_results, *map, keys_have_nulls, include_null_keys, stream);

  // Extract the populated indices from the hash map and create a gather map.
  // Gathering using this map from sparse results will give dense results.
  auto gather_map = extract_populated_keys(*map, keys.num_rows(), stream);

  // Compact all results from sparse_results and insert into cache
  sparse_to_dense_results(keys,
                          requests,
                          &sparse_results,
                          cache,
                          gather_map,
                          *map,
                          keys_have_nulls,
                          include_null_keys,
                          stream,
                          mr);

  return cudf::detail::gather(keys,
                              gather_map,
                              out_of_bounds_policy::DONT_CHECK,
                              cudf::detail::negative_index_policy::NOT_ALLOWED,
                              stream,
                              mr);
}

}  // namespace

/**
 * @brief Indicates if a set of aggregation requests can be satisfied with a
 * hash-based groupby implementation.
 *
 * @param requests The set of columns to aggregate and the aggregations to
 * perform
 * @return true A hash-based groupby should be used
 * @return false A hash-based groupby should not be used
 */
bool can_use_hash_groupby(host_span<aggregation_request const> requests)
{
  return std::all_of(requests.begin(), requests.end(), [](aggregation_request const& r) {
    // Currently, structs are not supported in any of hash-based aggregations.
    // Therefore, if any request contains structs then we must fallback to sort-based aggregations.
    // TODO: Support structs in hash-based aggregations.
    auto const v_type = is_dictionary(r.values.type())
                          ? cudf::dictionary_column_view(r.values).keys().type()
                          : r.values.type();

    return not(r.values.type().id() == type_id::STRUCT) and
           std::all_of(r.aggregations.begin(), r.aggregations.end(), [v_type](auto const& a) {
             return cudf::has_atomic_support(cudf::detail::target_type(v_type, a->kind)) and
                    is_hash_aggregation(a->kind);
           });
  });
}

// Hash-based groupby
std::pair<std::unique_ptr<table>, std::vector<aggregation_result>> groupby(
  table_view const& keys,
  host_span<aggregation_request const> requests,
  null_policy include_null_keys,
  rmm::cuda_stream_view stream,
  rmm::mr::device_memory_resource* mr)
{
  cudf::detail::result_cache cache(requests.size());

  std::unique_ptr<table> unique_keys =
    groupby(keys, requests, &cache, cudf::has_nulls(keys), include_null_keys, stream, mr);

  return std::pair(std::move(unique_keys), extract_results(requests, cache, stream, mr));
}
}  // namespace hash
}  // namespace detail
}  // namespace groupby
}  // namespace cudf
