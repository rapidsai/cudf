#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019-2021, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <groupby/common/utils.hpp>
#include <groupby/hash/groupby_kernels.cuh>

#include <cudf/aggregation.hpp>
#include <cudf/column/column.hpp>
#include <cudf/column/column_factories.hpp>
#include <cudf/column/column_view.hpp>
#include <cudf/copying.hpp>
#include <cudf/detail/aggregation/aggregation.cuh>
#include <cudf/detail/aggregation/aggregation.hpp>
#include <cudf/detail/aggregation/result_cache.hpp>
#include <cudf/detail/binaryop.hpp>
#include <cudf/detail/gather.cuh>
#include <cudf/detail/gather.hpp>
#include <cudf/detail/groupby.hpp>
#include <cudf/detail/replace.hpp>
#include <cudf/detail/unary.hpp>
#include <cudf/detail/utilities/cuda.cuh>
#include <cudf/detail/utilities/hash_functions.cuh>
#include <cudf/dictionary/dictionary_column_view.hpp>
#include <cudf/groupby.hpp>
#include <cudf/scalar/scalar.hpp>
#include <cudf/table/row_operators.cuh>
#include <cudf/table/table.hpp>
#include <cudf/table/table_device_view.cuh>
#include <cudf/table/table_view.hpp>
#include <cudf/types.hpp>
#include <cudf/utilities/traits.hpp>
#include <hash/concurrent_unordered_map.cuh>

#include <rmm/cuda_stream_view.hpp>

#include <memory>
#include <unordered_set>
#include <utility>

namespace cudf {
namespace groupby {
namespace detail {
namespace hash {
namespace {

/**
 * @brief List of aggregation operations that can be computed with a hash-based
 * implementation.
 */
constexpr std::array<aggregation::Kind, 12> hash_aggregations{aggregation::SUM,
                                                              aggregation::PRODUCT,
                                                              aggregation::MIN,
                                                              aggregation::MAX,
                                                              aggregation::COUNT_VALID,
                                                              aggregation::COUNT_ALL,
                                                              aggregation::ARGMIN,
                                                              aggregation::ARGMAX,
                                                              aggregation::SUM_OF_SQUARES,
                                                              aggregation::MEAN,
                                                              aggregation::STD,
                                                              aggregation::VARIANCE};

// Could be hash: SUM, PRODUCT, MIN, MAX, COUNT_VALID, COUNT_ALL, ANY, ALL,
// Compound: MEAN(SUM, COUNT_VALID), VARIANCE, STD(MEAN (SUM, COUNT_VALID), COUNT_VALID),
// ARGMAX, ARGMIN

// TODO replace with std::find in C++20 onwards.
template <class T, size_t N>
constexpr bool array_contains(std::array<T, N> const& haystack, T needle)
{
  for (auto const& val : haystack) {
    if (val == needle) return true;
  }
  return false;
}

/**
 * @brief Indicates whether the specified aggregation operation can be computed
 * with a hash-based implementation.
 *
 * @param t The aggregation operation to verify
 * @return true `t` is valid for a hash based groupby
 * @return false `t` is invalid for a hash based groupby
 */
bool constexpr is_hash_aggregation(aggregation::Kind t)
{
  return array_contains(hash_aggregations, t);
}

class groupby_simple_aggregations_collector final
  : public cudf::detail::simple_aggregations_collector {
 public:
  using cudf::detail::simple_aggregations_collector::visit;

  std::vector<std::unique_ptr<aggregation>> visit(data_type col_type,
                                                  cudf::detail::min_aggregation const&) override
  {
    std::vector<std::unique_ptr<aggregation>> aggs;
    aggs.push_back(col_type.id() == type_id::STRING ? make_argmin_aggregation()
                                                    : make_min_aggregation());
    return aggs;
  }

  std::vector<std::unique_ptr<aggregation>> visit(data_type col_type,
                                                  cudf::detail::max_aggregation const&) override
  {
    std::vector<std::unique_ptr<aggregation>> aggs;
    aggs.push_back(col_type.id() == type_id::STRING ? make_argmax_aggregation()
                                                    : make_max_aggregation());
    return aggs;
  }

  std::vector<std::unique_ptr<aggregation>> visit(data_type col_type,
                                                  cudf::detail::mean_aggregation const&) override
  {
    (void)col_type;
    CUDF_EXPECTS(is_fixed_width(col_type), "MEAN aggregation expects fixed width type");
    std::vector<std::unique_ptr<aggregation>> aggs;
    aggs.push_back(make_sum_aggregation());
    // COUNT_VALID
    aggs.push_back(make_count_aggregation());

    return aggs;
  }

  std::vector<std::unique_ptr<aggregation>> visit(data_type,
                                                  cudf::detail::var_aggregation const&) override
  {
    std::vector<std::unique_ptr<aggregation>> aggs;
    aggs.push_back(make_sum_aggregation());
    // COUNT_VALID
    aggs.push_back(make_count_aggregation());

    return aggs;
  }

  std::vector<std::unique_ptr<aggregation>> visit(data_type,
                                                  cudf::detail::std_aggregation const&) override
  {
    std::vector<std::unique_ptr<aggregation>> aggs;
    aggs.push_back(make_sum_aggregation());
    // COUNT_VALID
    aggs.push_back(make_count_aggregation());

    return aggs;
  }
};

template <typename Map>
class hash_compound_agg_finalizer final : public cudf::detail::aggregation_finalizer {
  size_t col_idx;
  column_view col;
  data_type result_type;
  cudf::detail::result_cache* sparse_results;
  cudf::detail::result_cache* dense_results;
  device_span<size_type const> gather_map;
  size_type const map_size;
  Map const& map;
  bitmask_type const* __restrict__ row_bitmask;
  rmm::mr::device_memory_resource* mr;
  rmm::cuda_stream_view stream;

 public:
  using cudf::detail::aggregation_finalizer::visit;

  hash_compound_agg_finalizer(size_t col_idx,
                              column_view col,
                              cudf::detail::result_cache* sparse_results,
                              cudf::detail::result_cache* dense_results,
                              device_span<size_type const> gather_map,
                              size_type map_size,
                              Map const& map,
                              bitmask_type const* row_bitmask,
                              rmm::cuda_stream_view stream,
                              rmm::mr::device_memory_resource* mr)
    : col_idx(col_idx),
      col(col),
      sparse_results(sparse_results),
      dense_results(dense_results),
      gather_map(gather_map),
      map_size(map_size),
      map(map),
      row_bitmask(row_bitmask),
      stream(stream),
      mr(mr)
  {
    result_type = cudf::is_dictionary(col.type()) ? cudf::dictionary_column_view(col).keys().type()
                                                  : col.type();
  }

  auto to_dense_agg_result(cudf::aggregation const& agg)
  {
    auto s                  = sparse_results->get_result(col_idx, agg);
    auto dense_result_table = cudf::detail::gather(table_view({std::move(s)}),
                                                   gather_map.begin(),
                                                   gather_map.begin() + map_size,
                                                   out_of_bounds_policy::DONT_CHECK,
                                                   stream,
                                                   mr);
    return std::move(dense_result_table->release()[0]);
  }

  // Enables conversion of ARGMIN/ARGMAX into MIN/MAX
  auto gather_argminmax(aggregation const& agg)
  {
    auto arg_result = to_dense_agg_result(agg);
    // We make a view of ARG(MIN/MAX) result without a null mask and gather
    // using this map. The values in data buffer of ARG(MIN/MAX) result
    // corresponding to null values was initialized to ARG(MIN/MAX)_SENTINEL
    // which is an out of bounds index value (-1) and causes the gathered
    // value to be null.
    column_view null_removed_map(
      data_type(type_to_id<size_type>()),
      arg_result->size(),
      static_cast<void const*>(arg_result->view().template data<size_type>()));
    auto gather_argminmax =
      cudf::detail::gather(table_view({col}),
                           null_removed_map,
                           arg_result->nullable() ? cudf::out_of_bounds_policy::NULLIFY
                                                  : cudf::out_of_bounds_policy::DONT_CHECK,
                           cudf::detail::negative_index_policy::NOT_ALLOWED,
                           stream,
                           mr);
    return std::move(gather_argminmax->release()[0]);
  }

  // Declare overloads for each kind of aggregation to dispatch
  void visit(cudf::aggregation const& agg) override
  {
    if (dense_results->has_result(col_idx, agg)) return;
    dense_results->add_result(col_idx, agg, to_dense_agg_result(agg));
  }

  void visit(cudf::detail::min_aggregation const& agg) override
  {
    if (dense_results->has_result(col_idx, agg)) return;
    if (result_type.id() == type_id::STRING) {
      auto transformed_agg = make_argmin_aggregation();
      dense_results->add_result(col_idx, agg, gather_argminmax(*transformed_agg));
    } else {
      dense_results->add_result(col_idx, agg, to_dense_agg_result(agg));
    }
  }

  void visit(cudf::detail::max_aggregation const& agg) override
  {
    if (dense_results->has_result(col_idx, agg)) return;

    if (result_type.id() == type_id::STRING) {
      auto transformed_agg = make_argmax_aggregation();
      dense_results->add_result(col_idx, agg, gather_argminmax(*transformed_agg));
    } else {
      dense_results->add_result(col_idx, agg, to_dense_agg_result(agg));
    }
  }

  void visit(cudf::detail::mean_aggregation const& agg) override
  {
    if (dense_results->has_result(col_idx, agg)) return;

    auto sum_agg   = make_sum_aggregation();
    auto count_agg = make_count_aggregation();
    this->visit(*sum_agg);
    this->visit(*count_agg);
    column_view sum_result   = dense_results->get_result(col_idx, *sum_agg);
    column_view count_result = dense_results->get_result(col_idx, *count_agg);

    auto result =
      cudf::detail::binary_operation(sum_result,
                                     count_result,
                                     binary_operator::DIV,
                                     cudf::detail::target_type(result_type, aggregation::MEAN),
                                     stream,
                                     mr);
    dense_results->add_result(col_idx, agg, std::move(result));
  }

  void visit(cudf::detail::var_aggregation const& agg) override
  {
    if (dense_results->has_result(col_idx, agg)) return;

    auto sum_agg   = make_sum_aggregation();
    auto count_agg = make_count_aggregation();
    this->visit(*sum_agg);
    this->visit(*count_agg);
    column_view sum_result   = sparse_results->get_result(col_idx, *sum_agg);
    column_view count_result = sparse_results->get_result(col_idx, *count_agg);

    auto values_view = column_device_view::create(col);
    auto sum_view    = column_device_view::create(sum_result);
    auto count_view  = column_device_view::create(count_result);

    auto var_result = make_fixed_width_column(
      cudf::detail::target_type(result_type, agg.kind), col.size(), mask_state::ALL_NULL, stream);
    auto var_result_view = mutable_column_device_view::create(var_result->mutable_view());
    mutable_table_view var_table_view{{var_result->mutable_view()}};
    cudf::detail::initialize_with_identity(var_table_view, {agg.kind}, stream);

    thrust::for_each_n(
      rmm::exec_policy(stream),
      thrust::make_counting_iterator(0),
      col.size(),
      ::cudf::detail::var_hash_functor<Map>{
        map, row_bitmask, *var_result_view, *values_view, *sum_view, *count_view, agg._ddof});
    sparse_results->add_result(col_idx, agg, std::move(var_result));
    dense_results->add_result(col_idx, agg, to_dense_agg_result(agg));
  }

  void visit(cudf::detail::std_aggregation const& agg) override
  {
    if (dense_results->has_result(col_idx, agg)) return;
    auto var_agg = make_variance_aggregation(agg._ddof);
    this->visit(*dynamic_cast<cudf::detail::var_aggregation*>(var_agg.get()));
    column_view variance = dense_results->get_result(col_idx, *var_agg);

    auto result = cudf::detail::unary_operation(variance, unary_operator::SQRT, stream, mr);
    dense_results->add_result(col_idx, agg, std::move(result));
  }
};
// flatten aggs to filter in single pass aggs
std::tuple<table_view,
           std::vector<aggregation::Kind>,
           std::vector<std::unique_ptr<aggregation>>,
           std::vector<size_t>>
flatten_single_pass_aggs(host_span<aggregation_request const> requests)
{
  std::vector<column_view> columns;
  std::vector<std::unique_ptr<aggregation>> aggs;
  std::vector<aggregation::Kind> agg_kinds;
  std::vector<size_t> col_ids;

  for (size_t i = 0; i < requests.size(); i++) {
    auto const& request = requests[i];
    auto const& agg_v   = request.aggregations;

    std::unordered_set<aggregation::Kind> agg_kinds_set;
    auto insert_agg =
      [&](size_t i, column_view const& request_values, std::unique_ptr<aggregation>&& agg) {
        if (agg_kinds_set.insert(agg->kind).second) {
          agg_kinds.push_back(agg->kind);
          aggs.push_back(std::move(agg));
          columns.push_back(request_values);
          col_ids.push_back(i);
        }
      };

    auto values_type = cudf::is_dictionary(request.values.type())
                         ? cudf::dictionary_column_view(request.values).keys().type()
                         : request.values.type();
    for (auto&& agg : agg_v) {
      groupby_simple_aggregations_collector collector;

      for (auto& agg_s : agg->get_simple_aggregations(values_type, collector)) {
        insert_agg(i, request.values, std::move(agg_s));
      }
    }
  }

  return std::make_tuple(
    table_view(columns), std::move(agg_kinds), std::move(aggs), std::move(col_ids));
}

/**
 * @brief Gather sparse results into dense using `gather_map` and add to
 * `dense_cache`
 *
 * @see groupby_null_templated()
 */
template <typename Map>
void sparse_to_dense_results(table_view const& keys,
                             host_span<aggregation_request const> requests,
                             cudf::detail::result_cache* sparse_results,
                             cudf::detail::result_cache* dense_results,
                             device_span<size_type const> gather_map,
                             size_type map_size,
                             Map const& map,
                             bool keys_have_nulls,
                             null_policy include_null_keys,
                             rmm::cuda_stream_view stream,
                             rmm::mr::device_memory_resource* mr)
{
  auto row_bitmask{bitmask_and(keys, stream, rmm::mr::get_current_device_resource())};
  bool skip_key_rows_with_nulls = keys_have_nulls and include_null_keys == null_policy::EXCLUDE;
  bitmask_type const* row_bitmask_ptr =
    skip_key_rows_with_nulls ? static_cast<bitmask_type*>(row_bitmask.data()) : nullptr;

  for (size_t i = 0; i < requests.size(); i++) {
    auto const& agg_v = requests[i].aggregations;
    auto const& col   = requests[i].values;

    // Given an aggregation, this will get the result from sparse_results and
    // convert and return dense, compacted result
    auto finalizer = hash_compound_agg_finalizer<Map>(i,
                                                      col,
                                                      sparse_results,
                                                      dense_results,
                                                      gather_map,
                                                      map_size,
                                                      map,
                                                      row_bitmask_ptr,
                                                      stream,
                                                      mr);
    for (auto&& agg : agg_v) {
      agg->finalize(finalizer);
    }
  }
}

/**
 * @brief Construct hash map that uses row comparator and row hasher on
 * `d_keys` table and stores indices
 */
template <bool keys_have_nulls>
auto create_hash_map(table_device_view const& d_keys,
                     null_policy include_null_keys,
                     rmm::cuda_stream_view stream)
{
  size_type constexpr unused_key{std::numeric_limits<size_type>::max()};
  size_type constexpr unused_value{std::numeric_limits<size_type>::max()};

  using map_type = concurrent_unordered_map<size_type,
                                            size_type,
                                            row_hasher<default_hash, keys_have_nulls>,
                                            row_equality_comparator<keys_have_nulls>>;

  using allocator_type = typename map_type::allocator_type;

  bool const null_keys_are_equal{include_null_keys == null_policy::INCLUDE};

  row_hasher<default_hash, keys_have_nulls> hasher{d_keys};
  row_equality_comparator<keys_have_nulls> rows_equal{d_keys, d_keys, null_keys_are_equal};

  return map_type::create(compute_hash_table_size(d_keys.num_rows()),
                          stream,
                          unused_key,
                          unused_value,
                          hasher,
                          rows_equal,
                          allocator_type());
}

// make table that will hold sparse results
auto create_sparse_results_table(table_view const& flattened_values,
                                 std::vector<aggregation::Kind> aggs,
                                 rmm::cuda_stream_view stream)
{
  // TODO single allocation - room for performance improvement
  std::vector<std::unique_ptr<column>> sparse_columns;
  std::transform(
    flattened_values.begin(),
    flattened_values.end(),
    aggs.begin(),
    std::back_inserter(sparse_columns),
    [stream](auto const& col, auto const& agg) {
      bool nullable =
        (agg == aggregation::COUNT_VALID or agg == aggregation::COUNT_ALL)
          ? false
          : (col.has_nulls() or agg == aggregation::VARIANCE or agg == aggregation::STD);
      auto mask_flag = (nullable) ? mask_state::ALL_NULL : mask_state::UNALLOCATED;

      auto col_type = cudf::is_dictionary(col.type())
                        ? cudf::dictionary_column_view(col).keys().type()
                        : col.type();

      return make_fixed_width_column(
        cudf::detail::target_type(col_type, agg), col.size(), mask_flag, stream);
    });

  table sparse_table(std::move(sparse_columns));
  mutable_table_view table_view = sparse_table.mutable_view();
  cudf::detail::initialize_with_identity(table_view, aggs, stream);
  return sparse_table;
}

/**
 * @brief Computes all aggregations from `requests` that require a single pass
 * over the data and stores the results in `sparse_results`
 *
 * @see groupby_null_templated()
 */
template <bool keys_have_nulls, typename Map>
void compute_single_pass_aggs(table_view const& keys,
                              host_span<aggregation_request const> requests,
                              cudf::detail::result_cache* sparse_results,
                              Map& map,
                              null_policy include_null_keys,
                              rmm::cuda_stream_view stream)
{
  // flatten the aggs to a table that can be operated on by aggregate_row
  auto const [flattened_values, agg_kinds, aggs, col_ids] = flatten_single_pass_aggs(requests);

  // make table that will hold sparse results
  table sparse_table = create_sparse_results_table(flattened_values, agg_kinds, stream);
  // prepare to launch kernel to do the actual aggregation
  auto d_sparse_table = mutable_table_device_view::create(sparse_table, stream);
  auto d_values       = table_device_view::create(flattened_values, stream);
  auto const d_aggs   = cudf::detail::make_device_uvector_async(agg_kinds, stream);

  bool skip_key_rows_with_nulls = keys_have_nulls and include_null_keys == null_policy::EXCLUDE;

  auto row_bitmask =
    skip_key_rows_with_nulls ? cudf::detail::bitmask_and(keys, stream) : rmm::device_buffer{};
  thrust::for_each_n(
    rmm::exec_policy(stream),
    thrust::make_counting_iterator(0),
    keys.num_rows(),
    hash::compute_single_pass_aggs_fn<Map>{map,
                                           keys.num_rows(),
                                           *d_values,
                                           *d_sparse_table,
                                           d_aggs.data(),
                                           static_cast<bitmask_type*>(row_bitmask.data()),
                                           skip_key_rows_with_nulls});
  // Add results back to sparse_results cache
  auto sparse_result_cols = sparse_table.release();
  for (size_t i = 0; i < aggs.size(); i++) {
    // Note that the cache will make a copy of this temporary aggregation
    sparse_results->add_result(col_ids[i], *aggs[i], std::move(sparse_result_cols[i]));
  }
}

/**
 * @brief Computes and returns a device vector containing all populated keys in
 * `map`.
 */
template <typename Map>
std::pair<rmm::device_uvector<size_type>, size_type> extract_populated_keys(
  Map map, size_type num_keys, rmm::cuda_stream_view stream)
{
  rmm::device_uvector<size_type> populated_keys(num_keys, stream);

  auto get_key = [] __device__(auto const& element) {
    size_type key, value;
    thrust::tie(key, value) = element;
    return key;
  };

  auto end_it = thrust::copy_if(
    rmm::exec_policy(stream),
    thrust::make_transform_iterator(map.data(), get_key),
    thrust::make_transform_iterator(map.data() + map.capacity(), get_key),
    populated_keys.begin(),
    [unused_key = map.get_unused_key()] __device__(size_type key) { return key != unused_key; });

  size_type map_size = end_it - populated_keys.begin();

  return std::make_pair(std::move(populated_keys), map_size);
}

/**
 * @brief Computes groupby using hash table.
 *
 * First, we create a hash table that stores the indices of unique rows in
 * `keys`. The upper limit on the number of values in this map is the number
 * of rows in `keys`.
 *
 * To store the results of aggregations, we create temporary sparse columns
 * which have the same size as input value columns. Using the hash map, we
 * determine the location within the sparse column to write the result of the
 * aggregation into.
 *
 * The sparse column results of all aggregations are stored into the cache
 * `sparse_results`. This enables the use of previously calculated results in
 * other aggregations.
 *
 * All the aggregations which can be computed in a single pass are computed
 * first, in a combined kernel. Then using these results, aggregations that
 * require multiple passes, will be computed.
 *
 * Finally, using the hash map, we generate a vector of indices of populated
 * values in sparse result columns. Then, for each aggregation originally
 * requested in `requests`, we gather sparse results into a column of dense
 * results using the aforementioned index vector. Dense results are stored into
 * the in/out parameter `cache`.
 */
template <bool keys_have_nulls>
std::unique_ptr<table> groupby_null_templated(table_view const& keys,
                                              host_span<aggregation_request const> requests,
                                              cudf::detail::result_cache* cache,
                                              null_policy include_null_keys,
                                              rmm::cuda_stream_view stream,
                                              rmm::mr::device_memory_resource* mr)
{
  auto d_keys = table_device_view::create(keys, stream);
  auto map    = create_hash_map<keys_have_nulls>(*d_keys, include_null_keys, stream);

  // Cache of sparse results where the location of aggregate value in each
  // column is indexed by the hash map
  cudf::detail::result_cache sparse_results(requests.size());

  // Compute all single pass aggs first
  compute_single_pass_aggs<keys_have_nulls>(
    keys, requests, &sparse_results, *map, include_null_keys, stream);

  // Extract the populated indices from the hash map and create a gather map.
  // Gathering using this map from sparse results will give dense results.
  auto map_and_size = extract_populated_keys(*map, keys.num_rows(), stream);
  rmm::device_uvector<size_type> gather_map{std::move(map_and_size.first)};
  size_type const map_size = map_and_size.second;

  // Compact all results from sparse_results and insert into cache
  sparse_to_dense_results(keys,
                          requests,
                          &sparse_results,
                          cache,
                          gather_map,
                          map_size,
                          *map,
                          keys_have_nulls,
                          include_null_keys,
                          stream,
                          mr);

  return cudf::detail::gather(keys,
                              gather_map.begin(),
                              gather_map.begin() + map_size,
                              out_of_bounds_policy::DONT_CHECK,
                              stream,
                              mr);
}

}  // namespace

/**
 * @brief Indicates if a set of aggregation requests can be satisfied with a
 * hash-based groupby implementation.
 *
 * @param keys The table of keys
 * @param requests The set of columns to aggregate and the aggregations to
 * perform
 * @return true A hash-based groupby should be used
 * @return false A hash-based groupby should not be used
 */
bool can_use_hash_groupby(table_view const& keys, host_span<aggregation_request const> requests)
{
  return std::all_of(requests.begin(), requests.end(), [](aggregation_request const& r) {
    return std::all_of(r.aggregations.begin(), r.aggregations.end(), [](auto const& a) {
      return is_hash_aggregation(a->kind);
    });
  });
}

// Hash-based groupby
std::pair<std::unique_ptr<table>, std::vector<aggregation_result>> groupby(
  table_view const& keys,
  host_span<aggregation_request const> requests,
  null_policy include_null_keys,
  rmm::cuda_stream_view stream,
  rmm::mr::device_memory_resource* mr)
{
  cudf::detail::result_cache cache(requests.size());

  std::unique_ptr<table> unique_keys;
  if (has_nulls(keys)) {
    unique_keys =
      groupby_null_templated<true>(keys, requests, &cache, include_null_keys, stream, mr);
  } else {
    unique_keys =
      groupby_null_templated<false>(keys, requests, &cache, include_null_keys, stream, mr);
  }

  return std::make_pair(std::move(unique_keys), extract_results(requests, cache));
}
}  // namespace hash
}  // namespace detail
}  // namespace groupby
}  // namespace cudf
