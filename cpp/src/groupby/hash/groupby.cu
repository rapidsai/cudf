/*
 * Copyright (c) 2019, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <cudf/column/column.hpp>
#include <cudf/column/column_view.hpp>
#include <cudf/groupby.hpp>
#include <cudf/detail/groupby.hpp>
#include <cudf/table/table.hpp>
#include <cudf/table/table_view.hpp>
#include <cudf/types.hpp>
#include <cudf/aggregation.hpp>
#include <cudf/detail/aggregation/aggregation.hpp>
#include <cudf/detail/aggregation/aggregation.cuh>

#include <memory>
#include <utility>

namespace cudf {
namespace experimental {
namespace groupby {
namespace detail {
namespace hash {
namespace {
// This is a temporary fix due to compiler bug and we can resort back to
// constexpr once cuda 10.2 becomes RAPIDS's minimum compiler version
#if 0
/**
 * @brief List of aggregation operations that can be computed with a hash-based
 * implementation.
 */
constexpr std::array<aggregation::Kind, 6> hash_aggregations{
    aggregation::SUM, aggregation::MIN, aggregation::MAX, 
    aggregation::COUNT_VALID, aggregation::COUNT_ALL, aggregation::MEAN};

template <class T, size_t N>
constexpr bool array_contains(std::array<T, N> const& haystack, T needle) {
  for (auto i = 0u; i < N; ++i) {
    if (haystack[i] == needle) return true;
  }
  return false;
}
#endif

/**
 * @brief Indicates whether the specified aggregation operation can be computed
 * with a hash-based implementation.
 *
 * @param t The aggregation operation to verify
 * @return true `t` is valid for a hash based groupby
 * @return false `t` is invalid for a hash based groupby
 */
bool constexpr is_hash_aggregation(aggregation::Kind t) {
  // this is a temporary fix due to compiler bug and we can resort back to
  // constexpr once cuda 10.2 becomes RAPIDS's minimum compiler version
  // return array_contains(hash_aggregations, t);
  return (t == aggregation::SUM) or (t == aggregation::MIN) or
         (t == aggregation::MAX) or (t == aggregation::COUNT_VALID) or
         (t == aggregation::COUNT_VALID) or (t == aggregation::MEAN);
}
}  // namespace

/**
 * @brief Indicates if a set of aggregation requests can be satisfied with a
 * hash-based groupby implementation.
 *
 * @param keys The table of keys
 * @param requests The set of columns to aggregate and the aggregations to
 * perform
 * @return true A hash-based groupby should be used
 * @return false A hash-based groupby should not be used
 */
bool can_use_hash_groupby(table_view const& keys,
                      std::vector<aggregation_request> const& requests) {
  return false;
  // TODO (dm): Jake to enable the following code when hash-groupby is ready
  // return std::all_of(
  //     requests.begin(), requests.end(), [](aggregation_request const& r) {
  //       return std::all_of(
  //           r.aggregations.begin(), r.aggregations.end(),
  //           [](auto const& a) { return is_hash_aggregation(a->kind); });
  //     });
}

// Hash-based groupby
std::pair<std::unique_ptr<table>, std::vector<aggregation_result>> groupby(
    table_view const& keys, std::vector<aggregation_request> const& requests,
    bool ignore_null_keys, hipStream_t stream,
    rmm::mr::device_memory_resource* mr) {
  // stub
  return std::make_pair(std::make_unique<table>(),
                        std::vector<aggregation_result>{});
}
}  // namespace hash
}  // namespace detail
}  // namespace groupby
}  // namespace experimental
}  // namespace cudf
