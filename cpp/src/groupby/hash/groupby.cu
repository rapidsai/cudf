#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019-2021, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <groupby/common/utils.hpp>
#include <groupby/hash/groupby_kernels.cuh>

#include <cudf/aggregation.hpp>
#include <cudf/column/column.hpp>
#include <cudf/column/column_factories.hpp>
#include <cudf/column/column_view.hpp>
#include <cudf/copying.hpp>
#include <cudf/detail/aggregation/aggregation.cuh>
#include <cudf/detail/aggregation/aggregation.hpp>
#include <cudf/detail/aggregation/result_cache.hpp>
#include <cudf/detail/binaryop.hpp>
#include <cudf/detail/gather.cuh>
#include <cudf/detail/gather.hpp>
#include <cudf/detail/groupby.hpp>
#include <cudf/detail/replace.hpp>
#include <cudf/detail/unary.hpp>
#include <cudf/detail/utilities/cuda.cuh>
#include <cudf/detail/utilities/hash_functions.cuh>
#include <cudf/dictionary/dictionary_column_view.hpp>
#include <cudf/groupby.hpp>
#include <cudf/scalar/scalar.hpp>
#include <cudf/table/row_operators.cuh>
#include <cudf/table/table.hpp>
#include <cudf/table/table_device_view.cuh>
#include <cudf/table/table_view.hpp>
#include <cudf/types.hpp>
#include <cudf/utilities/traits.hpp>
#include <hash/concurrent_unordered_map.cuh>

#include <rmm/cuda_stream_view.hpp>

#include <memory>
#include <unordered_set>
#include <utility>

namespace cudf {
namespace groupby {
namespace detail {
namespace hash {
namespace {
// This is a temporary fix due to compiler bug and we can resort back to
// constexpr once cuda 10.2 becomes RAPIDS's minimum compiler version
#if 0
/**
 * @brief List of aggregation operations that can be computed with a hash-based
 * implementation.
 */
constexpr std::array<aggregation::Kind, 10> hash_aggregations{
    aggregation::SUM, aggregation::MIN, aggregation::MAX,
    aggregation::COUNT_VALID, aggregation::COUNT_ALL,
    aggregation::ARGMIN, aggregation::ARGMAX,
    aggregation::SUM_OF_SQUARES,
    aggregation::MEAN, aggregation::STD, aggregation::VARIANCE};

//Could be hash: SUM, PRODUCT, MIN, MAX, COUNT_VALID, COUNT_ALL, ANY, ALL,
// Compound: MEAN(SUM, COUNT_VALID), VARIANCE, STD(MEAN (SUM, COUNT_VALID), COUNT_VALID),
// ARGMAX, ARGMIN
// FIXME(kn): adding SUM_OF_SQUARES causes ptxas compiler crash (<=CUDA 10.2) for more than 3 types!

template <class T, size_t N>
constexpr bool array_contains(std::array<T, N> const& haystack, T needle) {
  for (auto i = 0u; i < N; ++i) {
    if (haystack[i] == needle) return true;
  }
  return false;
}
#endif

/**
 * @brief Indicates whether the specified aggregation operation can be computed
 * with a hash-based implementation.
 *
 * @param t The aggregation operation to verify
 * @return true `t` is valid for a hash based groupby
 * @return false `t` is invalid for a hash based groupby
 */
bool constexpr is_hash_aggregation(aggregation::Kind t)
{
  // this is a temporary fix due to compiler bug and we can resort back to
  // constexpr once cuda 10.2 becomes RAPIDS's minimum compiler version
  // return array_contains(hash_aggregations, t);
  return (t == aggregation::SUM) or (t == aggregation::MIN) or (t == aggregation::MAX) or
         (t == aggregation::COUNT_VALID) or (t == aggregation::COUNT_ALL) or
         (t == aggregation::ARGMIN) or (t == aggregation::ARGMAX) or
         (t == aggregation::SUM_OF_SQUARES) or (t == aggregation::MEAN) or
         (t == aggregation::STD) or (t == aggregation::VARIANCE);
}

template <typename Map>
class hash_compound_agg_finalizer final : public cudf::detail::aggregation_finalizer {
  size_t col_idx;
  column_view col;
  data_type result_type;
  cudf::detail::result_cache* sparse_results;
  cudf::detail::result_cache* dense_results;
  rmm::device_vector<size_type> const& gather_map;
  size_type const map_size;
  Map const& map;
  bitmask_type const* __restrict__ row_bitmask;
  rmm::mr::device_memory_resource* mr;
  rmm::cuda_stream_view stream;

 public:
  hash_compound_agg_finalizer(size_t col_idx,
                              column_view col,
                              cudf::detail::result_cache* sparse_results,
                              cudf::detail::result_cache* dense_results,
                              rmm::device_vector<size_type> const& gather_map,
                              size_type map_size,
                              Map const& map,
                              bitmask_type const* row_bitmask,
                              rmm::cuda_stream_view stream,
                              rmm::mr::device_memory_resource* mr)
    : col_idx(col_idx),
      col(col),
      sparse_results(sparse_results),
      dense_results(dense_results),
      gather_map(gather_map),
      map_size(map_size),
      map(map),
      row_bitmask(row_bitmask),
      stream(stream),
      mr(mr)
  {
    result_type = cudf::is_dictionary(col.type()) ? cudf::dictionary_column_view(col).keys().type()
                                                  : col.type();
  }

  auto to_dense_agg_result(cudf::aggregation const& agg)
  {
    auto s                  = sparse_results->get_result(col_idx, agg);
    auto dense_result_table = cudf::detail::gather(table_view({s}),
                                                   gather_map.begin(),
                                                   gather_map.begin() + map_size,
                                                   out_of_bounds_policy::DONT_CHECK,
                                                   stream,
                                                   mr);
    return std::move(dense_result_table->release()[0]);
  }

  // Enables conversion of ARGMIN/ARGMAX into MIN/MAX
  auto gather_argminmax(aggregation::Kind const& agg_kind)
  {
    auto transformed_agg = std::make_unique<aggregation>(agg_kind);
    auto arg_result      = to_dense_agg_result(*transformed_agg);
    // We make a view of ARG(MIN/MAX) result without a null mask and gather
    // using this map. The values in data buffer of ARG(MIN/MAX) result
    // corresponding to null values was initialized to ARG(MIN/MAX)_SENTINEL
    // which is an out of bounds index value (-1) and causes the gathered
    // value to be null.
    column_view null_removed_map(
      data_type(type_to_id<size_type>()),
      arg_result->size(),
      static_cast<void const*>(arg_result->view().template data<size_type>()));
    auto gather_argminmax =
      cudf::detail::gather(table_view({col}),
                           null_removed_map,
                           arg_result->nullable() ? cudf::out_of_bounds_policy::NULLIFY
                                                  : cudf::out_of_bounds_policy::DONT_CHECK,
                           cudf::detail::negative_index_policy::NOT_ALLOWED,
                           stream,
                           mr);
    return std::move(gather_argminmax->release()[0]);
  };

  // Declare overloads for each kind of aggregation to dispatch
  void visit(cudf::aggregation const& agg) override
  {
    if (dense_results->has_result(col_idx, agg)) return;
    dense_results->add_result(col_idx, agg, to_dense_agg_result(agg));
  }

  void visit(cudf::detail::min_aggregation const& agg) override
  {
    if (dense_results->has_result(col_idx, agg)) return;
    if (result_type.id() == type_id::STRING)
      dense_results->add_result(col_idx, agg, gather_argminmax(aggregation::ARGMIN));
    else
      dense_results->add_result(col_idx, agg, to_dense_agg_result(agg));
  }

  void visit(cudf::detail::max_aggregation const& agg) override
  {
    if (dense_results->has_result(col_idx, agg)) return;

    if (result_type.id() == type_id::STRING)
      dense_results->add_result(col_idx, agg, gather_argminmax(aggregation::ARGMAX));
    else
      dense_results->add_result(col_idx, agg, to_dense_agg_result(agg));
  }

  void visit(cudf::detail::mean_aggregation const& agg) override
  {
    if (dense_results->has_result(col_idx, agg)) return;

    auto sum_agg   = make_sum_aggregation();
    auto count_agg = make_count_aggregation();
    this->visit(*sum_agg);
    this->visit(*count_agg);
    column_view sum_result   = dense_results->get_result(col_idx, *sum_agg);
    column_view count_result = dense_results->get_result(col_idx, *count_agg);

    auto result =
      cudf::detail::binary_operation(sum_result,
                                     count_result,
                                     binary_operator::DIV,
                                     cudf::detail::target_type(result_type, aggregation::MEAN),
                                     stream,
                                     mr);
    dense_results->add_result(col_idx, agg, std::move(result));
  }

  void visit(cudf::detail::var_aggregation const& agg) override
  {
    if (dense_results->has_result(col_idx, agg)) return;

    auto sum_agg   = make_sum_aggregation();
    auto count_agg = make_count_aggregation();
    this->visit(*sum_agg);
    this->visit(*count_agg);
    column_view sum_result   = sparse_results->get_result(col_idx, *sum_agg);
    column_view count_result = sparse_results->get_result(col_idx, *count_agg);

    auto values_view = column_device_view::create(col);
    auto sum_view    = column_device_view::create(sum_result);
    auto count_view  = column_device_view::create(count_result);

    auto var_result = make_fixed_width_column(
      cudf::detail::target_type(result_type, agg.kind), col.size(), mask_state::ALL_NULL, stream);
    auto var_result_view = mutable_column_device_view::create(var_result->mutable_view());
    mutable_table_view var_table_view{{var_result->mutable_view()}};
    cudf::detail::initialize_with_identity(var_table_view, {agg.kind}, stream);

    thrust::for_each_n(
      rmm::exec_policy(stream),
      thrust::make_counting_iterator(0),
      col.size(),
      ::cudf::detail::var_hash_functor<Map>{
        map, row_bitmask, *var_result_view, *values_view, *sum_view, *count_view, agg._ddof});
    sparse_results->add_result(col_idx, agg, std::move(var_result));
    dense_results->add_result(col_idx, agg, to_dense_agg_result(agg));
  }

  void visit(cudf::detail::std_aggregation const& agg) override
  {
    if (dense_results->has_result(col_idx, agg)) return;
    auto var_agg = make_variance_aggregation(agg._ddof);
    this->visit(*static_cast<cudf::detail::var_aggregation*>(var_agg.get()));
    column_view variance = dense_results->get_result(col_idx, *var_agg);

    auto result = cudf::detail::unary_operation(variance, unary_operator::SQRT, stream, mr);
    dense_results->add_result(col_idx, agg, std::move(result));
  }
};

// flatten aggs to filter in single pass aggs
std::tuple<table_view, std::vector<aggregation::Kind>, std::vector<size_t>>
flatten_single_pass_aggs(std::vector<aggregation_request> const& requests)
{
  std::vector<column_view> columns;
  std::vector<aggregation::Kind> agg_kinds;
  std::vector<size_t> col_ids;

  for (size_t i = 0; i < requests.size(); i++) {
    auto const& request = requests[i];
    auto const& agg_v   = request.aggregations;

    std::unordered_set<aggregation::Kind> agg_kinds_set;
    auto insert_agg = [&](size_t i, column_view const& request_values, aggregation::Kind k) {
      if (agg_kinds_set.insert(k).second) {
        agg_kinds.push_back(k);
        columns.push_back(request_values);
        col_ids.push_back(i);
      }
    };

    auto values_type = cudf::is_dictionary(request.values.type())
                         ? cudf::dictionary_column_view(request.values).keys().type()
                         : request.values.type();
    for (auto&& agg : agg_v) {
      for (auto const& agg_s : agg->get_simple_aggregations(values_type))
        insert_agg(i, request.values, agg_s);
    }
  }

  return std::make_tuple(table_view(columns), std::move(agg_kinds), std::move(col_ids));
}

/**
 * @brief Gather sparse results into dense using `gather_map` and add to
 * `dense_cache`
 *
 * @see groupby_null_templated()
 */
template <typename Map>
void sparse_to_dense_results(table_view const& keys,
                             std::vector<aggregation_request> const& requests,
                             cudf::detail::result_cache* sparse_results,
                             cudf::detail::result_cache* dense_results,
                             rmm::device_vector<size_type> const& gather_map,
                             size_type map_size,
                             Map const& map,
                             bool keys_have_nulls,
                             null_policy include_null_keys,
                             rmm::cuda_stream_view stream,
                             rmm::mr::device_memory_resource* mr)
{
  auto row_bitmask{bitmask_and(keys, stream, rmm::mr::get_current_device_resource())};
  bool skip_key_rows_with_nulls = keys_have_nulls and include_null_keys == null_policy::EXCLUDE;
  bitmask_type const* row_bitmask_ptr =
    skip_key_rows_with_nulls ? static_cast<bitmask_type*>(row_bitmask.data()) : nullptr;

  for (size_t i = 0; i < requests.size(); i++) {
    auto const& agg_v = requests[i].aggregations;
    auto const& col   = requests[i].values;

    // Given an aggregation, this will get the result from sparse_results and
    // convert and return dense, compacted result
    auto finalizer = hash_compound_agg_finalizer<Map>(i,
                                                      col,
                                                      sparse_results,
                                                      dense_results,
                                                      gather_map,
                                                      map_size,
                                                      map,
                                                      row_bitmask_ptr,
                                                      stream,
                                                      mr);
    for (auto&& agg : agg_v) { agg->finalize(finalizer); }
  }
}

/**
 * @brief Construct hash map that uses row comparator and row hasher on
 * `d_keys` table and stores indices
 */
template <bool keys_have_nulls>
auto create_hash_map(table_device_view const& d_keys,
                     null_policy include_null_keys,
                     rmm::cuda_stream_view stream)
{
  size_type constexpr unused_key{std::numeric_limits<size_type>::max()};
  size_type constexpr unused_value{std::numeric_limits<size_type>::max()};

  using map_type = concurrent_unordered_map<size_type,
                                            size_type,
                                            row_hasher<default_hash, keys_have_nulls>,
                                            row_equality_comparator<keys_have_nulls>>;

  using allocator_type = typename map_type::allocator_type;

  bool const null_keys_are_equal{include_null_keys == null_policy::INCLUDE};

  row_hasher<default_hash, keys_have_nulls> hasher{d_keys};
  row_equality_comparator<keys_have_nulls> rows_equal{d_keys, d_keys, null_keys_are_equal};

  return map_type::create(compute_hash_table_size(d_keys.num_rows()),
                          stream,
                          unused_key,
                          unused_value,
                          hasher,
                          rows_equal,
                          allocator_type());
}

// make table that will hold sparse results
auto create_sparse_results_table(table_view const& flattened_values,
                                 std::vector<aggregation::Kind> aggs,
                                 rmm::cuda_stream_view stream)
{
  // TODO single allocation - room for performance improvement
  std::vector<std::unique_ptr<column>> sparse_columns;
  std::transform(
    flattened_values.begin(),
    flattened_values.end(),
    aggs.begin(),
    std::back_inserter(sparse_columns),
    [stream](auto const& col, auto const& agg) {
      bool nullable =
        (agg == aggregation::COUNT_VALID or agg == aggregation::COUNT_ALL)
          ? false
          : (col.has_nulls() or agg == aggregation::VARIANCE or agg == aggregation::STD);
      auto mask_flag = (nullable) ? mask_state::ALL_NULL : mask_state::UNALLOCATED;

      auto col_type = cudf::is_dictionary(col.type())
                        ? cudf::dictionary_column_view(col).keys().type()
                        : col.type();

      return make_fixed_width_column(
        cudf::detail::target_type(col_type, agg), col.size(), mask_flag, stream);
    });

  table sparse_table(std::move(sparse_columns));
  mutable_table_view table_view = sparse_table.mutable_view();
  cudf::detail::initialize_with_identity(table_view, aggs, stream);
  return sparse_table;
}

/**
 * @brief Computes all aggregations from `requests` that require a single pass
 * over the data and stores the results in `sparse_results`
 *
 * @see groupby_null_templated()
 */
template <bool keys_have_nulls, typename Map>
void compute_single_pass_aggs(table_view const& keys,
                              std::vector<aggregation_request> const& requests,
                              cudf::detail::result_cache* sparse_results,
                              Map& map,
                              null_policy include_null_keys,
                              rmm::cuda_stream_view stream)
{
  // flatten the aggs to a table that can be operated on by aggregate_row
  table_view flattened_values;
  std::vector<aggregation::Kind> aggs;
  std::vector<size_t> col_ids;
  std::tie(flattened_values, aggs, col_ids) = flatten_single_pass_aggs(requests);

  // make table that will hold sparse results
  table sparse_table = create_sparse_results_table(flattened_values, aggs, stream);
  // prepare to launch kernel to do the actual aggregation
  auto d_sparse_table = mutable_table_device_view::create(sparse_table, stream);
  auto d_values       = table_device_view::create(flattened_values, stream);
  rmm::device_vector<aggregation::Kind> d_aggs(aggs);

  bool skip_key_rows_with_nulls = keys_have_nulls and include_null_keys == null_policy::EXCLUDE;

  auto row_bitmask =
    skip_key_rows_with_nulls ? cudf::detail::bitmask_and(keys, stream) : rmm::device_buffer{};
  thrust::for_each_n(
    rmm::exec_policy(stream),
    thrust::make_counting_iterator(0),
    keys.num_rows(),
    hash::compute_single_pass_aggs_fn<Map>{map,
                                           keys.num_rows(),
                                           *d_values,
                                           *d_sparse_table,
                                           d_aggs.data().get(),
                                           static_cast<bitmask_type*>(row_bitmask.data()),
                                           skip_key_rows_with_nulls});
  // Add results back to sparse_results cache
  auto sparse_result_cols = sparse_table.release();
  for (size_t i = 0; i < aggs.size(); i++) {
    // Note that the cache will make a copy of this temporary aggregation
    auto agg = std::make_unique<aggregation>(aggs[i]);
    sparse_results->add_result(col_ids[i], *agg, std::move(sparse_result_cols[i]));
  }
}

/**
 * @brief Computes and returns a device vector containing all populated keys in
 * `map`.
 */
template <typename Map>
std::pair<rmm::device_vector<size_type>, size_type> extract_populated_keys(
  Map map, size_type num_keys, rmm::cuda_stream_view stream)
{
  rmm::device_vector<size_type> populated_keys(num_keys);

  auto get_key = [] __device__(auto const& element) {
    size_type key, value;
    thrust::tie(key, value) = element;
    return key;
  };

  auto end_it = thrust::copy_if(
    rmm::exec_policy(stream),
    thrust::make_transform_iterator(map.data(), get_key),
    thrust::make_transform_iterator(map.data() + map.capacity(), get_key),
    populated_keys.begin(),
    [unused_key = map.get_unused_key()] __device__(size_type key) { return key != unused_key; });

  size_type map_size = end_it - populated_keys.begin();

  return std::make_pair(std::move(populated_keys), map_size);
}

/**
 * @brief Computes groupby using hash table.
 *
 * First, we create a hash table that stores the indices of unique rows in
 * `keys`. The upper limit on the number of values in this map is the number
 * of rows in `keys`.
 *
 * To store the results of aggregations, we create temporary sparse columns
 * which have the same size as input value columns. Using the hash map, we
 * determine the location within the sparse column to write the result of the
 * aggregation into.
 *
 * The sparse column results of all aggregations are stored into the cache
 * `sparse_results`. This enables the use of previously calculated results in
 * other aggregations.
 *
 * All the aggregations which can be computed in a single pass are computed
 * first, in a combined kernel. Then using these results, aggregations that
 * require multiple passes, will be computed.
 *
 * Finally, using the hash map, we generate a vector of indices of populated
 * values in sparse result columns. Then, for each aggregation originally
 * requested in `requests`, we gather sparse results into a column of dense
 * results using the aforementioned index vector. Dense results are stored into
 * the in/out parameter `cache`.
 */
template <bool keys_have_nulls>
std::unique_ptr<table> groupby_null_templated(table_view const& keys,
                                              std::vector<aggregation_request> const& requests,
                                              cudf::detail::result_cache* cache,
                                              null_policy include_null_keys,
                                              rmm::cuda_stream_view stream,
                                              rmm::mr::device_memory_resource* mr)
{
  auto d_keys = table_device_view::create(keys, stream);
  auto map    = create_hash_map<keys_have_nulls>(*d_keys, include_null_keys, stream);

  // Cache of sparse results where the location of aggregate value in each
  // column is indexed by the hash map
  cudf::detail::result_cache sparse_results(requests.size());

  // Compute all single pass aggs first
  compute_single_pass_aggs<keys_have_nulls>(
    keys, requests, &sparse_results, *map, include_null_keys, stream);

  // Extract the populated indices from the hash map and create a gather map.
  // Gathering using this map from sparse results will give dense results.
  rmm::device_vector<size_type> gather_map;
  size_type map_size;
  std::tie(gather_map, map_size) = extract_populated_keys(*map, keys.num_rows(), stream);

  // Compact all results from sparse_results and insert into cache
  sparse_to_dense_results(keys,
                          requests,
                          &sparse_results,
                          cache,
                          gather_map,
                          map_size,
                          *map,
                          keys_have_nulls,
                          include_null_keys,
                          stream,
                          mr);

  return cudf::detail::gather(keys,
                              gather_map.begin(),
                              gather_map.begin() + map_size,
                              out_of_bounds_policy::DONT_CHECK,
                              stream,
                              mr);
}

}  // namespace

/**
 * @brief Indicates if a set of aggregation requests can be satisfied with a
 * hash-based groupby implementation.
 *
 * @param keys The table of keys
 * @param requests The set of columns to aggregate and the aggregations to
 * perform
 * @return true A hash-based groupby should be used
 * @return false A hash-based groupby should not be used
 */
bool can_use_hash_groupby(table_view const& keys, std::vector<aggregation_request> const& requests)
{
  return std::all_of(requests.begin(), requests.end(), [](aggregation_request const& r) {
    return std::all_of(r.aggregations.begin(), r.aggregations.end(), [](auto const& a) {
      return is_hash_aggregation(a->kind);
    });
  });
}

// Hash-based groupby
std::pair<std::unique_ptr<table>, std::vector<aggregation_result>> groupby(
  table_view const& keys,
  std::vector<aggregation_request> const& requests,
  null_policy include_null_keys,
  rmm::cuda_stream_view stream,
  rmm::mr::device_memory_resource* mr)
{
  cudf::detail::result_cache cache(requests.size());

  std::unique_ptr<table> unique_keys;
  if (has_nulls(keys)) {
    unique_keys =
      groupby_null_templated<true>(keys, requests, &cache, include_null_keys, stream, mr);
  } else {
    unique_keys =
      groupby_null_templated<false>(keys, requests, &cache, include_null_keys, stream, mr);
  }

  return std::make_pair(std::move(unique_keys), extract_results(requests, cache));
}
}  // namespace hash
}  // namespace detail
}  // namespace groupby
}  // namespace cudf
