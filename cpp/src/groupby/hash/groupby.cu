#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019-20, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <groupby/common/utils.hpp>
#include <groupby/hash/groupby_kernels.cuh>

#include <cudf/aggregation.hpp>
#include <cudf/column/column.hpp>
#include <cudf/column/column_factories.hpp>
#include <cudf/column/column_view.hpp>
#include <cudf/detail/aggregation/aggregation.cuh>
#include <cudf/detail/aggregation/aggregation.hpp>
#include <cudf/detail/aggregation/result_cache.hpp>
#include <cudf/detail/gather.cuh>
#include <cudf/detail/gather.hpp>
#include <cudf/detail/groupby.hpp>
#include <cudf/detail/replace.hpp>
#include <cudf/detail/utilities/cuda.cuh>
#include <cudf/detail/utilities/hash_functions.cuh>
#include <cudf/groupby.hpp>
#include <cudf/scalar/scalar.hpp>
#include <cudf/table/row_operators.cuh>
#include <cudf/table/table.hpp>
#include <cudf/table/table_device_view.cuh>
#include <cudf/table/table_view.hpp>
#include <cudf/types.hpp>
#include <cudf/utilities/traits.hpp>
#include <hash/concurrent_unordered_map.cuh>

#include <memory>
#include <utility>

namespace cudf {
namespace groupby {
namespace detail {
namespace hash {
namespace {
// This is a temporary fix due to compiler bug and we can resort back to
// constexpr once cuda 10.2 becomes RAPIDS's minimum compiler version
#if 0
/**
 * @brief List of aggregation operations that can be computed with a hash-based
 * implementation.
 */
constexpr std::array<aggregation::Kind, 7> hash_aggregations{
    aggregation::SUM, aggregation::MIN, aggregation::MAX,
    aggregation::COUNT_VALID, aggregation::COUNT_ALL,
    aggregation::ARGMIN, aggregation::ARGMAX};

template <class T, size_t N>
constexpr bool array_contains(std::array<T, N> const& haystack, T needle) {
  for (auto i = 0u; i < N; ++i) {
    if (haystack[i] == needle) return true;
  }
  return false;
}
#endif

/**
 * @brief Indicates whether the specified aggregation operation can be computed
 * with a hash-based implementation.
 *
 * @param t The aggregation operation to verify
 * @return true `t` is valid for a hash based groupby
 * @return false `t` is invalid for a hash based groupby
 */
bool constexpr is_hash_aggregation(aggregation::Kind t)
{
  // this is a temporary fix due to compiler bug and we can resort back to
  // constexpr once cuda 10.2 becomes RAPIDS's minimum compiler version
  // return array_contains(hash_aggregations, t);
  return (t == aggregation::SUM) or (t == aggregation::MIN) or (t == aggregation::MAX) or
         (t == aggregation::COUNT_VALID) or (t == aggregation::COUNT_ALL) or
         (t == aggregation::ARGMIN) or (t == aggregation::ARGMAX);
}

// flatten aggs to filter in single pass aggs
std::tuple<table_view, std::vector<aggregation::Kind>, std::vector<size_t>>
flatten_single_pass_aggs(std::vector<aggregation_request> const& requests)
{
  std::vector<column_view> columns;
  std::vector<aggregation::Kind> agg_kinds;
  std::vector<size_t> col_ids;

  for (size_t i = 0; i < requests.size(); i++) {
    auto const& request = requests[i];
    auto const& agg_v   = request.aggregations;

    auto insert_agg = [&agg_kinds, &columns, &col_ids, &request, i](aggregation::Kind k) {
      agg_kinds.push_back(k);
      columns.push_back(request.values);
      col_ids.push_back(i);
    };

    for (auto&& agg : agg_v) {
      if (is_hash_aggregation(agg->kind)) {
        if (is_fixed_width(request.values.type()) or agg->kind == aggregation::COUNT_VALID or
            agg->kind == aggregation::COUNT_ALL) {
          insert_agg(agg->kind);
        } else if (request.values.type().id() == type_id::STRING) {
          // For string type, only ARGMIN, ARGMAX, MIN, and MAX are supported
          if (agg->kind == aggregation::ARGMIN or agg->kind == aggregation::ARGMAX) {
            insert_agg(agg->kind);
          } else if (agg->kind == aggregation::MIN) {
            insert_agg(aggregation::ARGMIN);
          } else if (agg->kind == aggregation::MAX) {
            insert_agg(aggregation::ARGMAX);
          }
        }
      }
    }
  }
  return std::make_tuple(table_view(columns), std::move(agg_kinds), std::move(col_ids));
}

/**
 * @brief Gather sparse results into dense using `gather_map` and add to
 * `dense_cache`
 *
 * @see groupby_null_templated()
 */
void sparse_to_dense_results(std::vector<aggregation_request> const& requests,
                             cudf::detail::result_cache const& sparse_results,
                             cudf::detail::result_cache* dense_results,
                             rmm::device_vector<size_type> const& gather_map,
                             size_type map_size,
                             hipStream_t stream,
                             rmm::mr::device_memory_resource* mr)
{
  for (size_t i = 0; i < requests.size(); i++) {
    auto const& agg_v = requests[i].aggregations;
    auto const& col   = requests[i].values;

    // Given an aggregation, this will get the result from sparse_results and
    // convert and return dense, compacted result
    auto to_dense_agg_result =
      [&sparse_results, &gather_map, map_size, i, mr, stream](auto const& agg) {
        auto s                  = sparse_results.get_result(i, agg);
        auto dense_result_table = cudf::detail::gather(
          table_view({s}), gather_map.begin(), gather_map.begin() + map_size, false, mr, stream);
        return std::move(dense_result_table->release()[0]);
      };

    // Enables conversion of ARGMIN/ARGMAX into MIN/MAX
    auto transformed_result = [&col, to_dense_agg_result, mr, stream](auto const& agg_kind) {
      auto transformed_agg = std::make_unique<aggregation>(agg_kind);
      auto arg_result      = to_dense_agg_result(*transformed_agg);
      // We make a view of ARG(MIN/MAX) result without a null mask and gather
      // using this map. The values in data buffer of ARG(MIN/MAX) result
      // corresponding to null values was initialized to ARG(MIN/MAX)_SENTINEL
      // which is an out of bounds index value (-1) and causes the gathered
      // value to be null.
      column_view null_removed_map(
        data_type(type_to_id<size_type>()),
        arg_result->size(),
        static_cast<void const*>(arg_result->view().template data<size_type>()));
      auto transformed_result =
        cudf::detail::gather(table_view({col}),
                             null_removed_map,
                             arg_result->nullable() ? cudf::detail::out_of_bounds_policy::IGNORE
                                                    : cudf::detail::out_of_bounds_policy::NULLIFY,
                             cudf::detail::negative_index_policy::NOT_ALLOWED,
                             mr,
                             stream);
      return std::move(transformed_result->release()[0]);
    };

    for (auto&& agg : agg_v) {
      auto const& agg_ref = *agg;
      if (agg->kind == aggregation::COUNT_VALID or agg->kind == aggregation::COUNT_ALL) {
        dense_results->add_result(i, agg_ref, to_dense_agg_result(agg_ref));
      } else if (col.type().id() == type_id::STRING and
                 (agg->kind == aggregation::MAX or agg->kind == aggregation::MIN)) {
        if (agg->kind == aggregation::MAX) {
          dense_results->add_result(i, agg_ref, transformed_result(aggregation::ARGMAX));
        } else if (agg->kind == aggregation::MIN) {
          dense_results->add_result(i, agg_ref, transformed_result(aggregation::ARGMIN));
        }
      } else if (sparse_results.has_result(i, agg_ref)) {
        dense_results->add_result(i, agg_ref, to_dense_agg_result(agg_ref));
      }
    }
  }
}

/**
 * @brief Construct hash map that uses row comparator and row hasher on
 * `d_keys` table and stores indices
 */
template <bool keys_have_nulls>
auto create_hash_map(table_device_view const& d_keys,
                     null_policy include_null_keys,
                     hipStream_t stream = 0)
{
  size_type constexpr unused_key{std::numeric_limits<size_type>::max()};
  size_type constexpr unused_value{std::numeric_limits<size_type>::max()};

  using map_type = concurrent_unordered_map<size_type,
                                            size_type,
                                            row_hasher<default_hash, keys_have_nulls>,
                                            row_equality_comparator<keys_have_nulls>>;

  using allocator_type = typename map_type::allocator_type;

  bool const null_keys_are_equal{include_null_keys == null_policy::INCLUDE};

  row_hasher<default_hash, keys_have_nulls> hasher{d_keys};
  row_equality_comparator<keys_have_nulls> rows_equal{d_keys, d_keys, null_keys_are_equal};

  return map_type::create(compute_hash_table_size(d_keys.num_rows()),
                          unused_key,
                          unused_value,
                          hasher,
                          rows_equal,
                          allocator_type(),
                          stream);
}

/**
 * @brief Computes all aggregations from `requests` that require a single pass
 * over the data and stores the results in `sparse_results`
 *
 * @see groupby_null_templated()
 */
template <bool keys_have_nulls, typename Map>
void compute_single_pass_aggs(table_view const& keys,
                              std::vector<aggregation_request> const& requests,
                              cudf::detail::result_cache* sparse_results,
                              Map& map,
                              null_policy include_null_keys,
                              hipStream_t stream)
{
  // flatten the aggs to a table that can be operated on by aggregate_row
  table_view flattened_values;
  std::vector<aggregation::Kind> aggs;
  std::vector<size_t> col_ids;
  std::tie(flattened_values, aggs, col_ids) = flatten_single_pass_aggs(requests);

  // make table that will hold sparse results
  std::vector<std::unique_ptr<column>> sparse_columns;
  std::transform(flattened_values.begin(),
                 flattened_values.end(),
                 aggs.begin(),
                 std::back_inserter(sparse_columns),
                 [stream](auto const& col, auto const& agg) {
                   bool nullable =
                     (agg == aggregation::COUNT_VALID or agg == aggregation::COUNT_ALL)
                       ? false
                       : col.has_nulls();
                   auto mask_flag = (nullable) ? mask_state::ALL_NULL : mask_state::UNALLOCATED;

                   return make_fixed_width_column(
                     cudf::detail::target_type(col.type(), agg), col.size(), mask_flag, stream);
                 });

  table sparse_table(std::move(sparse_columns));
  mutable_table_view table_view = sparse_table.mutable_view();
  cudf::detail::initialize_with_identity(table_view, aggs, stream);

  // prepare to launch kernel to do the actual aggregation
  auto d_sparse_table = mutable_table_device_view::create(sparse_table);
  auto d_values       = table_device_view::create(flattened_values);
  rmm::device_vector<aggregation::Kind> d_aggs(aggs);

  bool skip_key_rows_with_nulls = keys_have_nulls and include_null_keys == null_policy::EXCLUDE;

  if (skip_key_rows_with_nulls) {
    auto row_bitmask{bitmask_and(keys, rmm::mr::get_current_device_resource(), stream)};
    thrust::for_each_n(
      rmm::exec_policy(stream)->on(stream),
      thrust::make_counting_iterator(0),
      keys.num_rows(),
      hash::compute_single_pass_aggs<true, Map>{map,
                                                keys.num_rows(),
                                                *d_values,
                                                *d_sparse_table,
                                                d_aggs.data().get(),
                                                static_cast<bitmask_type*>(row_bitmask.data())});
  } else {
    thrust::for_each_n(
      rmm::exec_policy(stream)->on(stream),
      thrust::make_counting_iterator(0),
      keys.num_rows(),
      hash::compute_single_pass_aggs<false, Map>{
        map, keys.num_rows(), *d_values, *d_sparse_table, d_aggs.data().get(), nullptr});
  }

  // Add results back to sparse_results cache
  auto sparse_result_cols = sparse_table.release();
  for (size_t i = 0; i < aggs.size(); i++) {
    // Note that the cache will make a copy of this temporary aggregation
    auto agg = std::make_unique<aggregation>(aggs[i]);
    sparse_results->add_result(col_ids[i], *agg, std::move(sparse_result_cols[i]));
  }
}

/**
 * @brief Computes and returns a device vector containing all populated keys in
 * `map`.
 */
template <typename Map>
std::pair<rmm::device_vector<size_type>, size_type> extract_populated_keys(Map map,
                                                                           size_type num_keys,
                                                                           hipStream_t stream = 0)
{
  rmm::device_vector<size_type> populated_keys(num_keys);

  auto get_key = [] __device__(auto const& element) {
    size_type key, value;
    thrust::tie(key, value) = element;
    return key;
  };

  auto end_it = thrust::copy_if(
    rmm::exec_policy(stream)->on(stream),
    thrust::make_transform_iterator(map.data(), get_key),
    thrust::make_transform_iterator(map.data() + map.capacity(), get_key),
    populated_keys.begin(),
    [unused_key = map.get_unused_key()] __device__(size_type key) { return key != unused_key; });

  size_type map_size = end_it - populated_keys.begin();

  return std::make_pair(std::move(populated_keys), map_size);
}

/**
 * @brief Computes groupby using hash table.
 *
 * First, we create a hash table that stores the indices of unique rows in
 * `keys`. The upper limit on the number of values in this map is the number
 * of rows in `keys`.
 *
 * To store the results of aggregations, we create temporary sparse columns
 * which have the same size as input value columns. Using the hash map, we
 * determine the location within the sparse column to write the result of the
 * aggregation into.
 *
 * The sparse column results of all aggregations are stored into the cache
 * `sparse_results`. This enables the use of previously calculated results in
 * other aggregations.
 *
 * All the aggregations which can be computed in a single pass are computed
 * first, in a combined kernel. Then using these results, aggregations that
 * require multiple passes, will be computed.
 *
 * Finally, using the hash map, we generate a vector of indices of populated
 * values in sparse result columns. Then, for each aggregation originally
 * requested in `requests`, we gather sparse results into a column of dense
 * results using the aforementioned index vector. Dense results are stored into
 * the in/out parameter `cache`.
 *
 */
template <bool keys_have_nulls>
std::unique_ptr<table> groupby_null_templated(table_view const& keys,
                                              std::vector<aggregation_request> const& requests,
                                              cudf::detail::result_cache* cache,
                                              null_policy include_null_keys,
                                              hipStream_t stream,
                                              rmm::mr::device_memory_resource* mr)
{
  auto d_keys = table_device_view::create(keys);
  auto map    = create_hash_map<keys_have_nulls>(*d_keys, include_null_keys, stream);

  // Cache of sparse results where the location of aggregate value in each
  // column is indexed by the hash map
  cudf::detail::result_cache sparse_results(requests.size());

  // Compute all single pass aggs first
  compute_single_pass_aggs<keys_have_nulls>(
    keys, requests, &sparse_results, *map, include_null_keys, stream);

  // Now continue with remaining multi-pass aggs
  // <placeholder>

  // Extract the populated indices from the hash map and create a gather map.
  // Gathering using this map from sparse results will give dense results.
  rmm::device_vector<size_type> gather_map;
  size_type map_size;
  std::tie(gather_map, map_size) = extract_populated_keys(*map, keys.num_rows(), stream);

  // Compact all results from sparse_results and insert into cache
  sparse_to_dense_results(requests, sparse_results, cache, gather_map, map_size, stream, mr);

  return cudf::detail::gather(
    keys, gather_map.begin(), gather_map.begin() + map_size, false, mr, stream);
}

}  // namespace

/**
 * @brief Indicates if a set of aggregation requests can be satisfied with a
 * hash-based groupby implementation.
 *
 * @param keys The table of keys
 * @param requests The set of columns to aggregate and the aggregations to
 * perform
 * @return true A hash-based groupby should be used
 * @return false A hash-based groupby should not be used
 */
bool can_use_hash_groupby(table_view const& keys, std::vector<aggregation_request> const& requests)
{
  return std::all_of(requests.begin(), requests.end(), [](aggregation_request const& r) {
    return std::all_of(r.aggregations.begin(), r.aggregations.end(), [](auto const& a) {
      return is_hash_aggregation(a->kind);
    });
  });
}

// Hash-based groupby
std::pair<std::unique_ptr<table>, std::vector<aggregation_result>> groupby(
  table_view const& keys,
  std::vector<aggregation_request> const& requests,
  null_policy include_null_keys,
  hipStream_t stream,
  rmm::mr::device_memory_resource* mr)
{
  cudf::detail::result_cache cache(requests.size());

  std::unique_ptr<table> unique_keys;
  if (has_nulls(keys)) {
    unique_keys =
      groupby_null_templated<true>(keys, requests, &cache, include_null_keys, stream, mr);
  } else {
    unique_keys =
      groupby_null_templated<false>(keys, requests, &cache, include_null_keys, stream, mr);
  }

  return std::make_pair(std::move(unique_keys), extract_results(requests, cache));
}
}  // namespace hash
}  // namespace detail
}  // namespace groupby
}  // namespace cudf
