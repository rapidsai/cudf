/*
 * Copyright (c) 2019-2020, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "group_reductions.hpp"

#include <cudf/column/column_device_view.cuh>
#include <cudf/column/column_factories.hpp>
#include <cudf/column/column_view.hpp>
#include <cudf/detail/aggregation/aggregation.hpp>
#include <cudf/dictionary/detail/iterator.cuh>
#include <cudf/dictionary/dictionary_column_view.hpp>
#include <cudf/utilities/type_dispatcher.hpp>

#include <rmm/thrust_rmm_allocator.h>

#include <thrust/iterator/discard_iterator.h>
#include <thrust/reduce.h>

namespace cudf {
namespace groupby {
namespace detail {
namespace {

template <typename ResultType, typename Iterator>
struct var_transform {
  // column_device_view d_values;
  Iterator values_iter;
  ResultType const* d_means;
  size_type const* d_group_sizes;
  size_type const* d_group_labels;
  size_type ddof;

  __device__ ResultType operator()(size_type i)
  {
    // if (d_values.is_null(i)) return 0.0;
    if (!thrust::get<1>(values_iter[i])) return 0.0;

    // ResultType x         = d_values.element<T>(i);
    ResultType x = static_cast<ResultType>(thrust::get<0>(values_iter[i]));

    size_type group_idx  = d_group_labels[i];
    size_type group_size = d_group_sizes[group_idx];  //.element<size_type>(group_idx);

    // prevent divide by zero error
    if (group_size == 0 or group_size - ddof <= 0) return 0.0;

    ResultType mean = d_means[group_idx];  //.element<ResultType>(group_idx);
    return (x - mean) * (x - mean) / (group_size - ddof);
  }
};

template <typename ResultType, typename Iterator>
void reduce_by_key_fn(Iterator values_iter,
                      rmm::device_vector<size_type> const& group_labels,
                      ResultType const* d_means,
                      size_type const* d_group_sizes,
                      size_type ddof,
                      ResultType* d_result,
                      hipStream_t stream)
{
  auto var_iter = thrust::make_transform_iterator(
    thrust::make_counting_iterator(0),
    var_transform<ResultType, decltype(values_iter)>{
      values_iter, d_means, d_group_sizes, group_labels.data().get(), ddof});

  thrust::reduce_by_key(rmm::exec_policy(stream)->on(stream),
                        group_labels.begin(),
                        group_labels.end(),
                        var_iter,
                        thrust::make_discard_iterator(),
                        d_result);
}

struct var_functor {
  template <typename T>
  std::enable_if_t<std::is_arithmetic<T>::value, std::unique_ptr<column>> operator()(
    column_view const& values,
    column_view const& group_means,
    column_view const& group_sizes,
    rmm::device_vector<size_type> const& group_labels,
    size_type ddof,
    rmm::mr::device_memory_resource* mr,
    hipStream_t stream)
  {
// Running this in debug build causes a runtime error:
// `reduce_by_key failed on 2nd step: invalid device function`
#if !defined(__CUDACC_DEBUG__)
    using ResultType = cudf::detail::target_type_t<T, aggregation::Kind::VARIANCE>;

    std::unique_ptr<column> result = make_numeric_column(data_type(type_to_id<ResultType>()),
                                                         group_sizes.size(),
                                                         mask_state::UNINITIALIZED,
                                                         stream,
                                                         mr);

    auto values_view = column_device_view::create(values, stream);
    auto d_values    = *values_view;

    auto d_group_labels = group_labels.data().get();
    auto d_means        = group_means.data<ResultType>();
    auto d_group_sizes  = group_sizes.data<size_type>();
    auto d_result       = result->mutable_view().data<ResultType>();

    if (!cudf::is_dictionary(values.type())) {
      if (values.has_nulls()) {
        auto values_iter = d_values.pair_begin<T, true>();
        reduce_by_key_fn(values_iter, group_labels, d_means, d_group_sizes, ddof, d_result, stream);
      } else {
        auto values_iter = d_values.pair_begin<T, false>();
        reduce_by_key_fn(values_iter, group_labels, d_means, d_group_sizes, ddof, d_result, stream);
      }
    } else {  // dictionary column type uses special pair iterator
      if (values.has_nulls()) {
        auto values_iter =
          cudf::dictionary::detail::make_dictionary_pair_iterator<T, true>(*values_view);
        reduce_by_key_fn(values_iter, group_labels, d_means, d_group_sizes, ddof, d_result, stream);
      } else {
        auto values_iter =
          cudf::dictionary::detail::make_dictionary_pair_iterator<T, false>(*values_view);
        reduce_by_key_fn(values_iter, group_labels, d_means, d_group_sizes, ddof, d_result, stream);
      }
    }

    // set nulls
    auto result_view = mutable_column_device_view::create(*result, stream);
    thrust::for_each_n(rmm::exec_policy(stream)->on(stream),
                       thrust::make_counting_iterator(0),
                       group_sizes.size(),
                       [d_result = *result_view, d_group_sizes, ddof] __device__(size_type i) {
                         size_type group_size = d_group_sizes[i];
                         if (group_size == 0 or group_size - ddof <= 0)
                           d_result.set_null(i);
                         else
                           d_result.set_valid(i);
                       });

    return result;
#else
    CUDF_FAIL("Groupby std/var supported in debug build");
#endif
  }

  template <typename T, typename... Args>
  std::enable_if_t<!std::is_arithmetic<T>::value, std::unique_ptr<column>> operator()(
    Args&&... args)
  {
    CUDF_FAIL("Only numeric types are supported in std/variance");
  }
};

}  // namespace

std::unique_ptr<column> group_var(column_view const& values,
                                  column_view const& group_means,
                                  column_view const& group_sizes,
                                  rmm::device_vector<size_type> const& group_labels,
                                  size_type ddof,
                                  rmm::mr::device_memory_resource* mr,
                                  hipStream_t stream)
{
  auto values_type = cudf::is_dictionary(values.type())
                       ? dictionary_column_view(values).keys().type()
                       : values.type();

  return type_dispatcher(
    values_type, var_functor{}, values, group_means, group_sizes, group_labels, ddof, mr, stream);
}

}  // namespace detail
}  // namespace groupby
}  // namespace cudf
