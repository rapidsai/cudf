/*
 * Copyright (c) 2019-2020, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <groupby/common/utils.hpp>
#include "group_reductions.hpp"

#include <cudf/aggregation.hpp>
#include <cudf/column/column.hpp>
#include <cudf/column/column_factories.hpp>
#include <cudf/column/column_view.hpp>
#include <cudf/detail/aggregation/aggregation.hpp>
#include <cudf/detail/aggregation/result_cache.hpp>
#include <cudf/detail/binaryop.hpp>
#include <cudf/detail/gather.hpp>
#include <cudf/detail/groupby.hpp>
#include <cudf/detail/groupby/sort_helper.hpp>
#include <cudf/detail/unary.hpp>
#include <cudf/groupby.hpp>
#include <cudf/table/table.hpp>
#include <cudf/table/table_view.hpp>
#include <cudf/types.hpp>

#include <memory>
#include <unordered_map>
#include <utility>

namespace cudf {
namespace groupby {
namespace detail {
/**
 * @brief Functor to dispatch aggregation with
 *
 * This functor is to be used with `aggregation_dispatcher` to compute the
 * appropriate aggregation. If the values on which to run the aggregation are
 * unchanged, then this functor should be re-used. This is because it stores
 * memoised sorted and/or grouped values and re-using will save on computation
 * of these values.
 */
struct store_result_functor {
  store_result_functor(size_type col_idx,
                       column_view const& values,
                       sort::sort_groupby_helper& helper,
                       cudf::detail::result_cache& cache,
                       hipStream_t stream,
                       rmm::mr::device_memory_resource* mr)
    : col_idx(col_idx), values(values), helper(helper), cache(cache), stream(stream), mr(mr)
  {
  }

  template <aggregation::Kind k>
  void operator()(aggregation const& agg)
  {
  }

 private:
  /**
   * @brief Get the grouped values
   *
   * Computes the grouped values from @p values on first invocation and returns
   * the stored result on subsequent invocation
   */
  column_view get_grouped_values()
  {
    // TODO (dm): After implementing single pass multi-agg, explore making a
    //            cache of all grouped value columns rather than one at a time
    if (grouped_values)
      return grouped_values->view();
    else if (sorted_values)
      // TODO (dm): When we implement scan, it wouldn't be ok to return sorted
      //            values when asked for grouped values. Change this then.
      return sorted_values->view();
    else
      grouped_values = helper.grouped_values(values);
    return grouped_values->view();
  };

  /**
   * @brief Get the grouped and sorted values
   *
   * Computes the grouped and sorted (within each group) values from @p values
   * on first invocation and returns the stored result on subsequent invocation
   */
  column_view get_sorted_values()
  {
    if (not sorted_values) sorted_values = helper.sorted_values(values);
    return sorted_values->view();
  };

 private:
  size_type col_idx;                  ///< Index of column in requests being operated on
  sort::sort_groupby_helper& helper;  ///< Sort helper
  cudf::detail::result_cache& cache;  ///< cache of results to store into
  column_view const& values;          ///< Column of values to group and aggregate

  hipStream_t stream;                  ///< CUDA stream on which to execute kernels
  rmm::mr::device_memory_resource* mr;  ///< Memory resource to allocate space for results

  std::unique_ptr<column> sorted_values;   ///< Memoised grouped and sorted values
  std::unique_ptr<column> grouped_values;  ///< Memoised grouped values
};

template <>
void store_result_functor::operator()<aggregation::COUNT_VALID>(aggregation const& agg)
{
  if (cache.has_result(col_idx, agg)) return;

  cache.add_result(
    col_idx,
    agg,
    get_grouped_values().nullable()
      ? detail::group_count_valid(
          get_grouped_values(), helper.group_labels(), helper.num_groups(), mr, stream)
      : detail::group_count_all(helper.group_offsets(), helper.num_groups(), mr, stream));
}

template <>
void store_result_functor::operator()<aggregation::COUNT_ALL>(aggregation const& agg)
{
  if (cache.has_result(col_idx, agg)) return;

  cache.add_result(
    col_idx, agg, detail::group_count_all(helper.group_offsets(), helper.num_groups(), mr, stream));
}

template <>
void store_result_functor::operator()<aggregation::SUM>(aggregation const& agg)
{
  if (cache.has_result(col_idx, agg)) return;

  cache.add_result(col_idx,
                   agg,
                   detail::group_sum(
                     get_grouped_values(), helper.num_groups(), helper.group_labels(), mr, stream));
};

template <>
void store_result_functor::operator()<aggregation::ARGMAX>(aggregation const& agg)
{
  if (cache.has_result(col_idx, agg)) return;

  cache.add_result(col_idx,
                   agg,
                   detail::group_argmax(get_grouped_values(),
                                        helper.num_groups(),
                                        helper.group_labels(),
                                        helper.key_sort_order(),
                                        mr,
                                        stream));
};

template <>
void store_result_functor::operator()<aggregation::ARGMIN>(aggregation const& agg)
{
  if (cache.has_result(col_idx, agg)) return;

  cache.add_result(col_idx,
                   agg,
                   detail::group_argmin(get_grouped_values(),
                                        helper.num_groups(),
                                        helper.group_labels(),
                                        helper.key_sort_order(),
                                        mr,
                                        stream));
};

template <>
void store_result_functor::operator()<aggregation::MIN>(aggregation const& agg)
{
  if (cache.has_result(col_idx, agg)) return;

  auto result = [&]() {
    if (cudf::is_fixed_width(values.type())) {
      return detail::group_min(
        get_grouped_values(), helper.num_groups(), helper.group_labels(), mr, stream);
    } else {
      auto argmin_agg = make_argmin_aggregation();
      operator()<aggregation::ARGMIN>(*argmin_agg);
      column_view argmin_result = cache.get_result(col_idx, *argmin_agg);

      // We make a view of ARGMIN result without a null mask and gather using
      // this mask. The values in data buffer of ARGMIN result corresponding
      // to null values was initialized to ARGMIN_SENTINEL which is an out of
      // bounds index value and causes the gathered value to be null.
      column_view null_removed_map(
        data_type(type_to_id<size_type>()),
        argmin_result.size(),
        static_cast<void const*>(argmin_result.template data<size_type>()));
      auto transformed_result =
        cudf::detail::gather(table_view({values}),
                             null_removed_map,
                             argmin_result.nullable() ? cudf::detail::out_of_bounds_policy::IGNORE
                                                      : cudf::detail::out_of_bounds_policy::NULLIFY,
                             cudf::detail::negative_index_policy::NOT_ALLOWED,
                             mr,
                             stream);
      return std::move(transformed_result->release()[0]);
    }
  }();

  cache.add_result(col_idx, agg, std::move(result));
};

template <>
void store_result_functor::operator()<aggregation::MAX>(aggregation const& agg)
{
  if (cache.has_result(col_idx, agg)) return;

  auto result = [&]() {
    if (cudf::is_fixed_width(values.type())) {
      return detail::group_max(
        get_grouped_values(), helper.num_groups(), helper.group_labels(), mr, stream);
    } else {
      auto argmax_agg = make_argmax_aggregation();
      operator()<aggregation::ARGMAX>(*argmax_agg);
      column_view argmax_result = cache.get_result(col_idx, *argmax_agg);

      // We make a view of ARGMAX result without a null mask and gather using
      // this mask. The values in data buffer of ARGMAX result corresponding
      // to null values was initialized to ARGMAX_SENTINEL which is an out of
      // bounds index value and causes the gathered value to be null.
      column_view null_removed_map(
        data_type(type_to_id<size_type>()),
        argmax_result.size(),
        static_cast<void const*>(argmax_result.template data<size_type>()));
      auto transformed_result =
        cudf::detail::gather(table_view({values}),
                             null_removed_map,
                             argmax_result.nullable() ? cudf::detail::out_of_bounds_policy::IGNORE
                                                      : cudf::detail::out_of_bounds_policy::NULLIFY,
                             cudf::detail::negative_index_policy::NOT_ALLOWED,
                             mr,
                             stream);
      return std::move(transformed_result->release()[0]);
    }
  }();

  cache.add_result(col_idx, agg, std::move(result));
};

template <>
void store_result_functor::operator()<aggregation::MEAN>(aggregation const& agg)
{
  if (cache.has_result(col_idx, agg)) return;

  auto sum_agg   = make_sum_aggregation();
  auto count_agg = make_count_aggregation();
  operator()<aggregation::SUM>(*sum_agg);
  operator()<aggregation::COUNT_VALID>(*count_agg);
  column_view sum_result   = cache.get_result(col_idx, *sum_agg);
  column_view count_result = cache.get_result(col_idx, *count_agg);

  // TODO (dm): Special case for timestamp. Add target_type_impl for it.
  //            Blocked until we support operator+ on timestamps
  auto result =
    cudf::detail::binary_operation(sum_result,
                                   count_result,
                                   binary_operator::DIV,
                                   cudf::detail::target_type(values.type(), aggregation::MEAN),
                                   mr,
                                   stream);
  cache.add_result(col_idx, agg, std::move(result));
};

template <>
void store_result_functor::operator()<aggregation::VARIANCE>(aggregation const& agg)
{
  if (cache.has_result(col_idx, agg)) return;

  auto var_agg   = static_cast<cudf::detail::std_var_aggregation const&>(agg);
  auto mean_agg  = make_mean_aggregation();
  auto count_agg = make_count_aggregation();
  operator()<aggregation::MEAN>(*mean_agg);
  operator()<aggregation::COUNT_VALID>(*count_agg);
  column_view mean_result = cache.get_result(col_idx, *mean_agg);
  column_view group_sizes = cache.get_result(col_idx, *count_agg);

  auto result = detail::group_var(get_grouped_values(),
                                  mean_result,
                                  group_sizes,
                                  helper.group_labels(),
                                  var_agg._ddof,
                                  mr,
                                  stream);
  cache.add_result(col_idx, agg, std::move(result));
};

template <>
void store_result_functor::operator()<aggregation::STD>(aggregation const& agg)
{
  if (cache.has_result(col_idx, agg)) return;

  auto std_agg = static_cast<cudf::detail::std_var_aggregation const&>(agg);
  auto var_agg = make_variance_aggregation(std_agg._ddof);
  operator()<aggregation::VARIANCE>(*var_agg);
  column_view var_result = cache.get_result(col_idx, *var_agg);

  auto result = cudf::detail::unary_operation(var_result, unary_op::SQRT, mr, stream);
  cache.add_result(col_idx, agg, std::move(result));
};

template <>
void store_result_functor::operator()<aggregation::QUANTILE>(aggregation const& agg)
{
  if (cache.has_result(col_idx, agg)) return;

  auto count_agg = make_count_aggregation();
  operator()<aggregation::COUNT_VALID>(*count_agg);
  column_view group_sizes = cache.get_result(col_idx, *count_agg);
  auto quantile_agg       = static_cast<cudf::detail::quantile_aggregation const&>(agg);

  auto result = detail::group_quantiles(get_sorted_values(),
                                        group_sizes,
                                        helper.group_offsets(),
                                        helper.num_groups(),
                                        quantile_agg._quantiles,
                                        quantile_agg._interpolation,
                                        mr,
                                        stream);
  cache.add_result(col_idx, agg, std::move(result));
};

template <>
void store_result_functor::operator()<aggregation::MEDIAN>(aggregation const& agg)
{
  if (cache.has_result(col_idx, agg)) return;

  auto count_agg = make_count_aggregation();
  operator()<aggregation::COUNT_VALID>(*count_agg);
  column_view group_sizes = cache.get_result(col_idx, *count_agg);

  auto result = detail::group_quantiles(get_sorted_values(),
                                        group_sizes,
                                        helper.group_offsets(),
                                        helper.num_groups(),
                                        {0.5},
                                        interpolation::LINEAR,
                                        mr,
                                        stream);
  cache.add_result(col_idx, agg, std::move(result));
};

template <>
void store_result_functor::operator()<aggregation::NUNIQUE>(aggregation const& agg)
{
  if (cache.has_result(col_idx, agg)) return;

  auto nunique_agg = static_cast<cudf::detail::nunique_aggregation const&>(agg);

  auto result = detail::group_nunique(get_sorted_values(),
                                      helper.group_labels(),
                                      helper.num_groups(),
                                      helper.group_offsets(),
                                      nunique_agg._null_handling,
                                      mr,
                                      stream);
  cache.add_result(col_idx, agg, std::move(result));
};

template <>
void store_result_functor::operator()<aggregation::NTH_ELEMENT>(aggregation const& agg)
{
  if (cache.has_result(col_idx, agg)) return;

  auto nth_element_agg = static_cast<cudf::detail::nth_element_aggregation const&>(agg);

  auto count_agg = make_count_aggregation(nth_element_agg._null_handling);
  if (count_agg->kind == aggregation::COUNT_VALID)
    operator()<aggregation::COUNT_VALID>(*count_agg);
  else if (count_agg->kind == aggregation::COUNT_ALL)
    operator()<aggregation::COUNT_ALL>(*count_agg);
  else
    CUDF_FAIL("Wrong count aggregation kind");
  column_view group_sizes = cache.get_result(col_idx, *count_agg);

  cache.add_result(col_idx,
                   agg,
                   detail::group_nth_element(get_grouped_values(),
                                             group_sizes,
                                             helper.group_labels(),
                                             helper.group_offsets(),
                                             helper.num_groups(),
                                             nth_element_agg._n,
                                             nth_element_agg._null_handling,
                                             mr,
                                             stream));
}

template <>
void store_result_functor::operator()<aggregation::COLLECT>(aggregation const& agg)
{
  if (cache.has_result(col_idx, agg)) return;

  auto result = detail::group_collect(
    get_grouped_values(), helper.group_offsets(), helper.num_groups(), mr, stream);

  cache.add_result(col_idx, agg, std::move(result));
};

}  // namespace detail

// Sort-based groupby
std::pair<std::unique_ptr<table>, std::vector<aggregation_result>> groupby::sort_aggregate(
  std::vector<aggregation_request> const& requests,
  hipStream_t stream,
  rmm::mr::device_memory_resource* mr)
{
  // We're going to start by creating a cache of results so that aggs that
  // depend on other aggs will not have to be recalculated. e.g. mean depends on
  // sum and count. std depends on mean and count
  cudf::detail::result_cache cache(requests.size());

  for (size_t i = 0; i < requests.size(); i++) {
    auto store_functor =
      detail::store_result_functor(i, requests[i].values, helper(), cache, stream, mr);
    for (size_t j = 0; j < requests[i].aggregations.size(); j++) {
      // TODO (dm): single pass compute all supported reductions
      cudf::detail::aggregation_dispatcher(
        requests[i].aggregations[j]->kind, store_functor, *requests[i].aggregations[j]);
    }
  }

  auto results = detail::extract_results(requests, cache);

  return std::make_pair(helper().unique_keys(mr, stream), std::move(results));
}
}  // namespace groupby
}  // namespace cudf
