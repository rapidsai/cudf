/*
 * Copyright (c) 2019, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "result_cache.hpp"
#include "group_reductions.hpp"

#include <cudf/column/column.hpp>
#include <cudf/column/column_view.hpp>
#include <cudf/detail/groupby.hpp>
#include <cudf/detail/groupby/sort_helper.hpp>
#include <cudf/groupby.hpp>
#include <cudf/table/table.hpp>
#include <cudf/table/table_view.hpp>
#include <cudf/types.hpp>
#include <cudf/aggregation.hpp>
#include <cudf/detail/aggregation/aggregation.hpp>
#include <cudf/column/column_factories.hpp>
#include <cudf/detail/binaryop.hpp>
#include <cudf/detail/unary.hpp>

#include <memory>
#include <utility>
#include <unordered_map>

namespace cudf {
namespace experimental {
namespace groupby {
namespace detail {

/**
 * @brief Functor to dispatch aggregation with
 * 
 * This functor is to be used with `aggregation_dispatcher` to compute the
 * appropriate aggregation. If the values on which to run the aggregation are
 * unchanged, then this functor should be re-used. This is because it stores
 * memoised sorted and/or grouped values and re-using will save on computation
 * of these values.
 */
struct store_result_functor {

  store_result_functor(
    size_type col_idx,
    column_view const& values,
    sort::sort_groupby_helper & helper,
    result_cache & cache,
    hipStream_t stream,
    rmm::mr::device_memory_resource* mr)
  : col_idx(col_idx),
    values(values),
    helper(helper),
    cache(cache),
    stream(stream),
    mr(mr)
  {}

  template <aggregation::Kind k>
  void operator()(std::unique_ptr<aggregation> const& agg) {}

 private:

  /**
   * @brief Get the grouped values
   * 
   * Computes the grouped values from @p values on first invocation and returns
   * the stored result on subsequent invocation
   */
  column_view get_grouped_values() {
    // TODO (dm): After implementing single pass mutli-agg, explore making a
    //            cache of all grouped value columns rather than one at a time
    if (grouped_values)
      return grouped_values->view();
    else if (sorted_values)
      // TODO (dm): When we implement scan, it wouldn't be ok to return sorted
      //            values when asked for grouped values. Change this then.
      return sorted_values->view();
    else
      grouped_values = helper.grouped_values(values);
    return grouped_values->view();
  };

  /**
   * @brief Get the grouped and sorted values
   * 
   * Computes the grouped and sorted (within each group) values from @p values 
   * on first invocation and returns the stored result on subsequent invocation
   */
  column_view get_sorted_values() {
    if (not sorted_values)
      sorted_values = helper.sorted_values(values);
    return sorted_values->view();
  };

 private:
  size_type col_idx; ///< Index of column in requests being operated on
  sort::sort_groupby_helper & helper; ///< Sort helper
  result_cache & cache; ///< cache of results to store into
  column_view const& values; ///< Column of values to group and aggregate

  hipStream_t stream; ///< CUDA stream on which to execute kernels 
  rmm::mr::device_memory_resource* mr; ///< Memory resource to allocate space for results

  std::unique_ptr<column> sorted_values; ///< Memoised grouped and sorted values
  std::unique_ptr<column> grouped_values; ///< Memoised grouped values
};

template <>
void store_result_functor::operator()<aggregation::COUNT>(
  std::unique_ptr<aggregation> const& agg)
{
  if (cache.has_result(col_idx, agg))
    return;

  cache.add_result(col_idx, agg, 
                  detail::group_count(get_grouped_values(), 
                            helper.group_labels(),
                            helper.num_groups(), mr, stream));
}

template <>
void store_result_functor::operator()<aggregation::SUM>(
  std::unique_ptr<aggregation> const& agg)
{
  if (cache.has_result(col_idx, agg))
    return;

  auto count_agg = make_count_aggregation();
  operator()<aggregation::COUNT>(count_agg);
  column_view count_result = cache.get_result(col_idx, count_agg);

  cache.add_result(col_idx, agg, 
                  detail::group_sum(get_grouped_values(), count_result, 
                                    helper.group_labels(),
                                    mr, stream));
};

template <>
void store_result_functor::operator()<aggregation::MIN>(
  std::unique_ptr<aggregation> const& agg)
{
  if (cache.has_result(col_idx, agg))
    return;

  auto count_agg = make_count_aggregation();
  operator()<aggregation::COUNT>(count_agg);
  column_view count_result = cache.get_result(col_idx, count_agg);

  cache.add_result(col_idx, agg, 
                  detail::group_min(get_grouped_values(), count_result, 
                                    helper.group_labels(),
                                    mr, stream));
};

template <>
void store_result_functor::operator()<aggregation::MAX>(
  std::unique_ptr<aggregation> const& agg)
{
  if (cache.has_result(col_idx, agg))
    return;

  auto count_agg = make_count_aggregation();
  operator()<aggregation::COUNT>(count_agg);
  column_view count_result = cache.get_result(col_idx, count_agg);

  cache.add_result(col_idx, agg, 
                  detail::group_max(get_grouped_values(), count_result, 
                                    helper.group_labels(),
                                    mr, stream));
};

template <>
void store_result_functor::operator()<aggregation::MEAN>(
  std::unique_ptr<aggregation> const& agg)
{
  if (cache.has_result(col_idx, agg))
    return;

  auto sum_agg = make_sum_aggregation();
  auto count_agg = make_count_aggregation();
  operator()<aggregation::SUM>(sum_agg);
  operator()<aggregation::COUNT>(count_agg);
  column_view sum_result = cache.get_result(col_idx, sum_agg);
  column_view count_result = cache.get_result(col_idx, count_agg);

  // TODO (dm): Special case for timestamp. Add target_type_impl for it.
  //            Blocked until we support operator+ on timestamps
  auto result = cudf::experimental::detail::binary_operation(
    sum_result, count_result, binary_operator::DIV, 
    cudf::experimental::detail::target_type(
      values.type(), aggregation::MEAN),
    mr, stream);
  cache.add_result(col_idx, agg, std::move(result));
};

template <>
void store_result_functor::operator()<aggregation::VARIANCE>(
  std::unique_ptr<aggregation> const& agg)
{
  if (cache.has_result(col_idx, agg))
    return;

  auto var_agg =
    static_cast<experimental::detail::std_var_aggregation const*>(agg.get());
  auto mean_agg = make_mean_aggregation();
  auto count_agg = make_count_aggregation();
  operator()<aggregation::MEAN>(mean_agg);
  operator()<aggregation::COUNT>(count_agg);
  column_view mean_result = cache.get_result(col_idx, mean_agg);
  column_view group_sizes = cache.get_result(col_idx, count_agg);

  auto result = detail::group_var(get_grouped_values(), mean_result, 
                          group_sizes, helper.group_labels(),
                          var_agg->_ddof, mr, stream);
  cache.add_result(col_idx, agg, std::move(result));
};

template <>
void store_result_functor::operator()<aggregation::STD>(
  std::unique_ptr<aggregation> const& agg)
{
  if (cache.has_result(col_idx, agg))
    return;

  auto std_agg =
    static_cast<experimental::detail::std_var_aggregation const*>(agg.get());
  auto var_agg = make_variance_aggregation(std_agg->_ddof);
  operator()<aggregation::VARIANCE>(var_agg);
  column_view var_result = cache.get_result(col_idx, var_agg);

  auto result = experimental::detail::unary_operation(
    var_result, experimental::unary_op::SQRT, mr, stream);
  cache.add_result(col_idx, agg, std::move(result));
};

template <>
void store_result_functor::operator()<aggregation::QUANTILE>(
  std::unique_ptr<aggregation> const& agg)
{
  if (cache.has_result(col_idx, agg))
    return;

  auto count_agg = make_count_aggregation();
  operator()<aggregation::COUNT>(count_agg);
  column_view group_sizes = cache.get_result(col_idx, count_agg);
  auto quantile_agg =
    static_cast<experimental::detail::quantile_aggregation const*>(agg.get());

  auto result = detail::group_quantiles(
    get_sorted_values(), group_sizes, helper.group_offsets(),
    quantile_agg->_quantiles, quantile_agg->_interpolation, mr, stream);
  cache.add_result(col_idx, agg, std::move(result));
};

template <>
void store_result_functor::operator()<aggregation::MEDIAN>(
  std::unique_ptr<aggregation> const& agg)
{
  if (cache.has_result(col_idx, agg))
    return;

  auto count_agg = make_count_aggregation();
  operator()<aggregation::COUNT>(count_agg);
  column_view group_sizes = cache.get_result(col_idx, count_agg);

  auto result = detail::group_quantiles(
    get_sorted_values(), group_sizes, helper.group_offsets(),
    {0.5}, interpolation::LINEAR, mr, stream);
  cache.add_result(col_idx, agg, std::move(result));
};


std::vector<aggregation_result> extract_results(
    std::vector<aggregation_request> const& requests,
    result_cache& cache)
{
  std::vector<aggregation_result> results(requests.size());

  for (size_t i = 0; i < requests.size(); i++) {
    for (auto &&agg : requests[i].aggregations) {
      results[i].results.emplace_back( cache.release_result(i, agg) );      
    }
  }
  return results;
}

}  // namespace detail

// Sort-based groupby
std::pair<std::unique_ptr<table>, std::vector<aggregation_result>> 
groupby::sort_aggregate(
    std::vector<aggregation_request> const& requests,
    hipStream_t stream, rmm::mr::device_memory_resource* mr)
{
  // We're going to start by creating a cache of results so that aggs that
  // depend on other aggs will not have to be recalculated. e.g. mean depends on
  // sum and count. std depends on mean and count
  detail::result_cache cache(requests.size());
  
  for (size_t i = 0; i < requests.size(); i++) {
    auto store_functor = detail::store_result_functor(i, requests[i].values,
                                    helper(), cache, stream, mr);
    for (size_t j = 0; j < requests[i].aggregations.size(); j++) {
      // TODO (dm): single pass compute all supported reductions
      experimental::detail::aggregation_dispatcher(
        requests[i].aggregations[j]->kind,
        store_functor,
        requests[i].aggregations[j]);
    }
  }  
  
  auto results = extract_results(requests, cache);
  
  return std::make_pair(helper().unique_keys(mr, stream),
                        std::move(results));
}
}  // namespace groupby
}  // namespace experimental
}  // namespace cudf
