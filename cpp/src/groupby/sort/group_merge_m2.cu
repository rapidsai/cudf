/*
 * Copyright (c) 2021-2025, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <cudf/column/column_factories.hpp>
#include <cudf/detail/aggregation/aggregation.hpp>
#include <cudf/detail/valid_if.cuh>
#include <cudf/structs/structs_column_view.hpp>
#include <cudf/utilities/span.hpp>
#include <cudf/utilities/type_dispatcher.hpp>

#include <rmm/cuda_stream_view.hpp>
#include <rmm/exec_policy.hpp>

#include <cuda/std/functional>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/iterator/zip_iterator.h>
#include <thrust/transform.h>
#include <thrust/tuple.h>

namespace cudf {
namespace groupby {
namespace detail {
namespace {

using result_type = double;
static_assert(
  std::is_same_v<cudf::detail::target_type_t<result_type, aggregation::Kind::M2>, result_type>);

/**
 * @brief Functor to merge partial results of `COUNT_VALID`, `MEAN`, and `M2` aggregations
 * for a given group (key) index.
 */
template <typename count_type>
struct merge_fn {
  size_type const* d_offsets;
  count_type const* d_counts;
  result_type const* d_means;
  result_type const* d_M2s;

  auto __device__ operator()(size_type const group_idx) const
  {
    count_type n{0};
    result_type avg{0};
    result_type m2{0};

    auto const start_idx = d_offsets[group_idx], end_idx = d_offsets[group_idx + 1];
    for (auto idx = start_idx; idx < end_idx; ++idx) {
      auto const partial_n = d_counts[idx];
      if (partial_n == 0) { continue; }
      auto const partial_avg = d_means[idx];
      auto const partial_m2  = d_M2s[idx];
      auto const new_n       = n + partial_n;
      auto const delta       = partial_avg - avg;
      m2 += partial_m2 + delta * delta * n * partial_n / new_n;
      avg = (avg * n + partial_avg * partial_n) / new_n;
      n   = new_n;
    }

    // If there are all nulls in the partial results (i.e., sum of all valid counts is
    // zero), then the output is a null.
    auto const is_valid = n > 0;
    return thrust::tuple{n, avg, m2, is_valid};
  }
};

template <typename count_type>
std::unique_ptr<column> merge_m2(column_view const& values,
                                 cudf::device_span<size_type const> group_offsets,
                                 size_type num_groups,
                                 rmm::cuda_stream_view stream,
                                 rmm::device_async_resource_ref mr)
{
  auto result_counts = make_numeric_column(
    data_type(type_to_id<count_type>()), num_groups, mask_state::UNALLOCATED, stream, mr);
  auto result_means = make_numeric_column(
    data_type(type_to_id<result_type>()), num_groups, mask_state::UNALLOCATED, stream, mr);
  auto result_M2s = make_numeric_column(
    data_type(type_to_id<result_type>()), num_groups, mask_state::UNALLOCATED, stream, mr);
  auto validities = rmm::device_uvector<bool>(num_groups, stream);

  // Perform merging for all the aggregations. Their output (and their validity data) are written
  // out concurrently through an output zip iterator.
  auto const out_iter =
    thrust::make_zip_iterator(result_counts->mutable_view().template data<count_type>(),
                              result_means->mutable_view().template data<result_type>(),
                              result_M2s->mutable_view().template data<result_type>(),
                              validities.begin());

  auto const count_valid = values.child(0);
  auto const mean_values = values.child(1);
  auto const M2_values   = values.child(2);
  auto const iter        = thrust::make_counting_iterator<size_type>(0);

  auto const fn = merge_fn<count_type>{group_offsets.begin(),
                                       count_valid.template begin<count_type>(),
                                       mean_values.template begin<result_type>(),
                                       M2_values.template begin<result_type>()};
  thrust::transform(rmm::exec_policy_nosync(stream), iter, iter + num_groups, out_iter, fn);

  // Generate bitmask for the output.
  // Only mean and M2 values can be nullable. Count column must be non-nullable.
  auto [null_mask, null_count] =
    cudf::detail::valid_if(validities.begin(), validities.end(), cuda::std::identity{}, stream, mr);
  if (null_count > 0) {
    result_means->set_null_mask(null_mask, null_count, stream);   // copy null_mask
    result_M2s->set_null_mask(std::move(null_mask), null_count);  // take over null_mask
  }

  // Output is a structs column containing the merged values of `COUNT_VALID`, `MEAN`, and `M2`.
  std::vector<std::unique_ptr<column>> out_columns;
  out_columns.emplace_back(std::move(result_counts));
  out_columns.emplace_back(std::move(result_means));
  out_columns.emplace_back(std::move(result_M2s));
  return cudf::make_structs_column(
    num_groups, std::move(out_columns), 0, rmm::device_buffer{0, stream, mr}, stream, mr);
}

}  // namespace

std::unique_ptr<column> group_merge_m2(column_view const& values,
                                       cudf::device_span<size_type const> group_offsets,
                                       size_type num_groups,
                                       rmm::cuda_stream_view stream,
                                       rmm::device_async_resource_ref mr)
{
  CUDF_EXPECTS(values.type().id() == type_id::STRUCT,
               "Input to `group_merge_m2` must be a structs column.");
  CUDF_EXPECTS(values.num_children() == 3,
               "Input to `group_merge_m2` must be a structs column having 3 children columns.");

  // The input column stores tuples of values (`COUNT_VALID`, `MEAN`, and `M2`).
  // However, the data type for `COUNT_VALID` must be wide enough such as
  // `INT64` or `FLOAT64` to prevent overflow when summing up.
  // For Apache Spark, the data type used for storing this is `FLOAT64`.
  auto const count_type_id = values.child(0).type().id();
  CUDF_EXPECTS((count_type_id == type_id::INT64 || count_type_id == type_id::FLOAT64) &&
                 values.child(1).type().id() == type_to_id<result_type>() &&
                 values.child(2).type().id() == type_to_id<result_type>(),
               "Input to `group_merge_m2` has invalid children type.");

  return count_type_id == type_id::INT64
           ? merge_m2<int64_t>(values, group_offsets, num_groups, stream, mr)
           : merge_m2<result_type>(values, group_offsets, num_groups, stream, mr);
}

}  // namespace detail
}  // namespace groupby
}  // namespace cudf
