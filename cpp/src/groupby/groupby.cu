/*
 * Copyright (c) 2019, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <cudf/column/column.hpp>
#include <cudf/column/column_factories.hpp>
#include <cudf/column/column_view.hpp>
#include <cudf/copying.hpp>
#include <cudf/detail/aggregation/aggregation.hpp>
#include <cudf/detail/gather.hpp>
#include <cudf/detail/groupby.hpp>
#include <cudf/detail/groupby/sort_helper.hpp>
#include <cudf/detail/nvtx/ranges.hpp>
#include <cudf/groupby.hpp>
#include <cudf/table/table.hpp>
#include <cudf/table/table_view.hpp>
#include <cudf/types.hpp>
#include <cudf/utilities/error.hpp>

#include <rmm/cuda_stream_view.hpp>

#include <thrust/copy.h>

#include <memory>
#include <utility>

namespace cudf {
namespace groupby {
// Constructor
groupby::groupby(table_view const& keys,
                 null_policy include_null_keys,
                 sorted keys_are_sorted,
                 std::vector<order> const& column_order,
                 std::vector<null_order> const& null_precedence)
  : _keys{keys},
    _include_null_keys{include_null_keys},
    _keys_are_sorted{keys_are_sorted},
    _column_order{column_order},
    _null_precedence{null_precedence}
{
}

// Select hash vs. sort groupby implementation
std::pair<std::unique_ptr<table>, std::vector<aggregation_result>> groupby::dispatch_aggregation(
  std::vector<aggregation_request> const& requests,
  hipStream_t stream,
  rmm::mr::device_memory_resource* mr)
{
  // If sort groupby has been called once on this groupby object, then
  // always use sort groupby from now on. Because once keys are sorted,
  // all the aggs that can be done by hash groupby are efficiently done by
  // sort groupby as well.
  // Only use hash groupby if the keys aren't sorted and all requests can be
  // satisfied with a hash implementation
  if (_keys_are_sorted == sorted::NO and not _helper and
      detail::hash::can_use_hash_groupby(_keys, requests)) {
    return detail::hash::groupby(_keys, requests, _include_null_keys, stream, mr);
  } else {
    return sort_aggregate(requests, stream, mr);
  }
}

// Destructor
// Needs to be in source file because sort_groupby_helper was forward declared
groupby::~groupby() = default;

namespace {
/// Make an empty table with appropriate types for requested aggs
auto empty_results(std::vector<aggregation_request> const& requests)
{
  std::vector<aggregation_result> empty_results;

  std::transform(
    requests.begin(), requests.end(), std::back_inserter(empty_results), [](auto const& request) {
      std::vector<std::unique_ptr<column>> results;

      std::transform(
        request.aggregations.begin(),
        request.aggregations.end(),
        std::back_inserter(results),
        [&request](auto const& agg) {
          return make_empty_column(cudf::detail::target_type(request.values.type(), agg->kind));
        });

      return aggregation_result{std::move(results)};
    });

  return empty_results;
}

/// Verifies the agg requested on the request's values is valid
void verify_valid_requests(std::vector<aggregation_request> const& requests)
{
  CUDF_EXPECTS(std::all_of(requests.begin(),
                           requests.end(),
                           [](auto const& request) {
                             return std::all_of(request.aggregations.begin(),
                                                request.aggregations.end(),
                                                [&request](auto const& agg) {
                                                  return cudf::detail::is_valid_aggregation(
                                                    request.values.type(), agg->kind);
                                                });
                           }),
               "Invalid type/aggregation combination.");
}

}  // namespace

// Compute aggregation requests
std::pair<std::unique_ptr<table>, std::vector<aggregation_result>> groupby::aggregate(
  std::vector<aggregation_request> const& requests, rmm::mr::device_memory_resource* mr)
{
  CUDF_FUNC_RANGE();
  CUDF_EXPECTS(
    std::all_of(requests.begin(),
                requests.end(),
                [this](auto const& request) { return request.values.size() == _keys.num_rows(); }),
    "Size mismatch between request values and groupby keys.");

  verify_valid_requests(requests);

  if (_keys.num_rows() == 0) { return std::make_pair(empty_like(_keys), empty_results(requests)); }

  return dispatch_aggregation(requests, 0, mr);
}

groupby::groups groupby::get_groups(table_view values, rmm::mr::device_memory_resource* mr)
{
  CUDF_FUNC_RANGE();
  auto grouped_keys = helper().sorted_keys(rmm::cuda_stream_default, mr);

  auto group_offsets = helper().group_offsets(0);
  std::vector<size_type> group_offsets_vector(group_offsets.size());
  thrust::copy(group_offsets.begin(), group_offsets.end(), group_offsets_vector.begin());

  std::unique_ptr<table> grouped_values{nullptr};
  if (values.num_columns()) {
    grouped_values = cudf::detail::gather(values,
                                          helper().key_sort_order(),
                                          cudf::detail::out_of_bounds_policy::NULLIFY,
                                          cudf::detail::negative_index_policy::NOT_ALLOWED,
                                          rmm::cuda_stream_default,
                                          mr);
    return groupby::groups{
      std::move(grouped_keys), std::move(group_offsets_vector), std::move(grouped_values)};
  } else {
    return groupby::groups{std::move(grouped_keys), std::move(group_offsets_vector)};
  }
}

// Get the sort helper object
detail::sort::sort_groupby_helper& groupby::helper()
{
  if (_helper) return *_helper;
  _helper = std::make_unique<detail::sort::sort_groupby_helper>(
    _keys, _include_null_keys, _keys_are_sorted);
  return *_helper;
};

}  // namespace groupby
}  // namespace cudf
