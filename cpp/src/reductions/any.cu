#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */
// The translation unit for reduction `max`

#include <cudf/detail/reduction_functions.hpp>
#include "simple.cuh"

std::unique_ptr<cudf::scalar> cudf::reduction::any(column_view const& col,
                                                   cudf::data_type const output_dtype,
                                                   rmm::mr::device_memory_resource* mr,
                                                   hipStream_t stream)
{
  CUDF_EXPECTS(output_dtype == cudf::data_type(cudf::type_id::BOOL8),
               "any() operation can be applied with output type `bool8` only");
  return cudf::reduction::max(col, cudf::data_type(cudf::type_id::BOOL8), mr, stream);
}
