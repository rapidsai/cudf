/*
 * Copyright (c) 2019-2020, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <cudf/column/column_factories.hpp>
#include <cudf/detail/copy.hpp>
#include <cudf/detail/indexalator.cuh>
#include <cudf/detail/reduction_functions.hpp>
#include <cudf/detail/unary.hpp>
#include <cudf/dictionary/dictionary_column_view.hpp>
#include <reductions/simple.cuh>

namespace cudf {
namespace reduction {
namespace {

// TODO: This may become the new result_type_dispatche
template <typename Op>
struct same_type_dispatcher {
  template <typename T, typename std::enable_if_t<std::is_floating_point<T>::value>* = nullptr>
  std::unique_ptr<scalar> operator()(column_view const& col,
                                     rmm::mr::device_memory_resource* mr,
                                     hipStream_t stream)
  {
    if (cudf::is_dictionary(col.type()))
      return simple::dictionary_reduction<T, double, Op>(col, mr, stream);
    return cudf::reduction::simple::simple_reduction<T, double, Op>(
      col, cudf::data_type{cudf::type_to_id<double>()}, mr, stream);
  }

  template <typename T, typename std::enable_if_t<std::is_integral<T>::value>* = nullptr>
  std::unique_ptr<scalar> operator()(column_view const& col,
                                     rmm::mr::device_memory_resource* mr,
                                     hipStream_t stream)
  {
    if (cudf::is_dictionary(col.type()))
      return simple::dictionary_reduction<T, int64_t, Op>(col, mr, stream);
    return cudf::reduction::simple::simple_reduction<T, int64_t, Op>(
      col, cudf::data_type{cudf::type_to_id<int64_t>()}, mr, stream);
  }

  template <typename T,
            typename std::enable_if_t<!std::is_floating_point<T>::value and
                                      !std::is_integral<T>::value>* = nullptr>
  std::unique_ptr<scalar> operator()(column_view const& col,
                                     rmm::mr::device_memory_resource* mr,
                                     hipStream_t stream)
  {
    CUDF_FAIL("Reduction operator not supported for this type");
  }
};

}  // namespace

std::unique_ptr<cudf::scalar> sum_of_squares(column_view const& col,
                                             cudf::data_type const output_dtype,
                                             rmm::mr::device_memory_resource* mr,
                                             hipStream_t stream)
{
  using reducer = same_type_dispatcher<cudf::reduction::op::sum_of_squares>;

  auto col_type =
    cudf::is_dictionary(col.type()) ? dictionary_column_view(col).keys().type() : col.type();

  auto result = cudf::type_dispatcher(col_type, reducer(), col, mr, stream);

  if (output_dtype == result->type() || !result->is_valid(stream)) return result;

  // if the output_dtype does not match, do extra work to cast it here
  auto input = cudf::make_column_from_scalar(*result, 1, mr, stream);
  // should build a scalar cast function
  auto output = cudf::detail::cast(*input, output_dtype, mr, stream);
  return cudf::detail::get_element(*output, 0, stream, mr);
}

}  // namespace reduction
}  // namespace cudf
