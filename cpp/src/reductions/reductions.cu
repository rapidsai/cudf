#include "hip/hip_runtime.h"
#include "cudf.h"
#include "utilities/cudf_utils.h"
#include "utilities/error_utils.h"
#include "utilities/type_dispatcher.hpp"

#include <cub/block/block_reduce.cuh>

#include <limits>
#include <type_traits>

#define REDUCTION_BLOCK_SIZE 128

struct IdentityLoader{
    template<typename T>
    __device__
    T operator() (const T *ptr, int pos) const {
        return ptr[pos];
    }
};

/*
Generic reduction implementation with support for validity mask
*/

template<typename T, typename F, typename Ld>
__global__
void gpu_reduction_op(const T *data, const gdf_valid_type *mask,
                      gdf_size_type size, T *results, F functor, T identity,
                      Ld loader)
{
    typedef hipcub::BlockReduce<T, REDUCTION_BLOCK_SIZE> BlockReduce;
    __shared__ typename BlockReduce::TempStorage temp_storage;

    int tid = threadIdx.x;
    int blkid = blockIdx.x;
    int blksz = blockDim.x;
    int gridsz = gridDim.x;

    int step = blksz * gridsz;

    T agg = identity;

    for (int base=blkid * blksz; base<size; base+=step) {
        // Threadblock synchronous loop
        int i = base + tid;
        // load
        T loaded = identity;
        if (i < size && gdf_is_valid(mask, i))
            loaded = loader(data, i);
            
        // Block reduce
        T temp = BlockReduce(temp_storage).Reduce(loaded, functor);
        // Add current block
        agg = functor(agg, temp);
    }
    // First thread of each block stores the result.
    if (tid == 0)
        results[blkid] = agg;
}



template<typename T, typename F>
struct ReduceOp {
    static
    gdf_error launch(gdf_column *input, T identity, T *output,
                     gdf_size_type output_size) {

        // 1st round
        //    Partially reduce the input into *output_size* length.
        //    Each block computes one output in *output*.
        //    output_size == gridsize
        typedef typename F::Loader Ld1;
        F functor1;
        Ld1 loader1;
        launch_once((const T*)input->data, input->valid, input->size,
                    (T*)output, output_size, identity, functor1, loader1);
        CUDA_CHECK_LAST();

        // 2nd round
        //    Finish the partial reduction (if needed).
        //    A single block reduction that computes one output stored to the
        //    first index in *output*.
        if ( output_size > 1 ) {
            typedef typename F::second F2;
            typedef typename F2::Loader Ld2;
            F2 functor2;
            Ld2 loader2;

            launch_once(output, nullptr, output_size,
                        output, 1, identity, functor2, loader2);
            CUDA_CHECK_LAST();
        }

        return GDF_SUCCESS;
    }

    template <typename Functor, typename Loader>
    static
    void launch_once(const T *data, gdf_valid_type *valid, gdf_size_type size,
                     T *output, gdf_size_type output_size, T identity,
                     Functor functor, Loader loader) {
        // find needed gridsize
        // use atmost REDUCTION_BLOCK_SIZE blocks
        int blocksize = REDUCTION_BLOCK_SIZE;
        int gridsize = (output_size < REDUCTION_BLOCK_SIZE?
                        output_size : REDUCTION_BLOCK_SIZE);

        // launch kernel
        gpu_reduction_op<<<gridsize, blocksize>>>(
            // inputs
            data, valid, size,
            // output
            output,
            // action
            functor,
            // identity
            identity,
            // loader
            loader
        );
    }

};


struct DeviceSum {
    typedef IdentityLoader Loader;
    typedef DeviceSum second;

    template<typename T>
    __device__
    T operator() (const T &lhs, const T &rhs) {
        return lhs + rhs;
    }

    template<typename T>
    static constexpr T identity() { return T{0}; }
};

struct DeviceProduct {
    typedef IdentityLoader Loader;
    typedef DeviceProduct second;

    template<typename T>
    __device__
    T operator() (const T &lhs, const T &rhs) {
        return lhs * rhs;
    }

    template<typename T>
    static constexpr T identity() { return T{1}; }
};

struct DeviceSumOfSquares {
    struct Loader {
        template<typename T>
        __device__
        T operator() (const T* ptr, int pos) const {
            T val = ptr[pos];   // load
            return val * val;   // squared
        }
    };
    // round 2 just uses the basic sum reduction
    typedef DeviceSum second;

    template<typename T>
    __device__
    T operator() (const T &lhs, const T &rhs) const {
        return lhs + rhs;
    }

    template<typename T>
    static constexpr T identity() { return T{0}; }
};

struct DeviceMin {
    typedef IdentityLoader Loader;
    typedef DeviceMin second;

    template<typename T>
    __device__
    T operator() (const T &lhs, const T &rhs) {
        return lhs <= rhs? lhs: rhs;
    }

    template<typename T>
    static constexpr T identity() { return std::numeric_limits<T>::max(); }
};

struct DeviceMax {
    typedef IdentityLoader Loader;
    typedef DeviceMax second;

    template<typename T>
    __device__
    T operator() (const T &lhs, const T &rhs) {
        return lhs >= rhs? lhs: rhs;
    }

    template<typename T>
    static constexpr T identity() { return std::numeric_limits<T>::lowest(); }
};

template <typename Op>
struct ReduceDispatcher {
    template <typename T,
              typename std::enable_if_t<std::is_arithmetic<T>::value>* = nullptr>
    gdf_error operator()(gdf_column *col, 
                         void *dev_result, 
                         gdf_size_type dev_result_size) {
        T identity = Op::template identity<T>();
        return ReduceOp<T, Op>::launch(col, identity, 
                                       reinterpret_cast<T*>(dev_result), 
                                       dev_result_size); 
    }

    template <typename T,
              typename std::enable_if_t<!std::is_arithmetic<T>::value, T>* = nullptr>
    gdf_error operator()(gdf_column *col, 
                         void *dev_result, 
                         gdf_size_type dev_result_size) {
        return GDF_UNSUPPORTED_DTYPE;
    }
};


gdf_error gdf_sum(gdf_column *col,
                  void *dev_result,
                  gdf_size_type dev_result_size)
{   
    return cudf::type_dispatcher(col->dtype, ReduceDispatcher<DeviceSum>(),
                                 col, dev_result, dev_result_size);
}

gdf_error gdf_product(gdf_column *col,
                      void *dev_result,
                      gdf_size_type dev_result_size)
{
    return cudf::type_dispatcher(col->dtype, ReduceDispatcher<DeviceProduct>(),
                                 col, dev_result, dev_result_size);
}

gdf_error gdf_sum_of_squares(gdf_column *col,
                             void *dev_result,
                             gdf_size_type dev_result_size)
{
    return cudf::type_dispatcher(col->dtype, ReduceDispatcher<DeviceSumOfSquares>(),
                                 col, dev_result, dev_result_size);
}

gdf_error gdf_min(gdf_column *col,
                  void *dev_result,
                  gdf_size_type dev_result_size)
{
    return cudf::type_dispatcher(col->dtype, ReduceDispatcher<DeviceMin>(),
                                 col, dev_result, dev_result_size);
}

gdf_error gdf_max(gdf_column *col,
                  void *dev_result,
                  gdf_size_type dev_result_size)
{
    return cudf::type_dispatcher(col->dtype, ReduceDispatcher<DeviceMax>(),
                                 col, dev_result, dev_result_size);
}


unsigned int gdf_reduce_optimal_output_size() {
    return REDUCTION_BLOCK_SIZE;
}
