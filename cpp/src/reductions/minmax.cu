#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2020, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */
// The translation unit for reduction `minmax`

#include <thrust/transform_reduce.h>
#include <cudf/column/column_device_view.cuh>
#include <cudf/column/column_view.hpp>
#include <cudf/detail/utilities/cuda.cuh>
#include <cudf/detail/utilities/device_operators.cuh>
#include <cudf/reduction.hpp>
#include <cudf/scalar/scalar_factories.hpp>
#include <type_traits>

namespace cudf {
namespace detail {

namespace {

/**
 * @brief Reduce the binary operation in device and return a device scalar.
 *
 * @tparam Op Binary operator functor
 * @tparam InputIterator Input iterator Type
 * @param d_in input iterator
 * @param num_items number of items to reduce
 * @param binary_op binary operator used to reduce
 * @param mr Device resource used for result allocation
 * @param stream CUDA stream to run kernels on.
 * @return rmm::device_scalar<OutputType>
 */
template <typename Op,
          typename InputIterator,
          typename OutputType = typename thrust::iterator_value<InputIterator>::type>
// typename std::enable_if_t<is_fixed_width<OutputType>()>* = nullptr>
rmm::device_scalar<OutputType> reduce_device(InputIterator d_in,
                                             cudf::size_type num_items,
                                             Op binary_op,
                                             rmm::mr::device_memory_resource *mr,
                                             hipStream_t stream)
{
  OutputType identity{};
  rmm::device_scalar<OutputType> dev_result{identity, stream, mr};  // TODO remove mr

  // Allocate temporary storage
  rmm::device_buffer d_temp_storage;
  size_t temp_storage_bytes = 0;
  hipcub::DeviceReduce::Reduce(d_temp_storage.data(),
                            temp_storage_bytes,
                            d_in,
                            dev_result.data(),
                            num_items,
                            binary_op,
                            identity,
                            stream);
  d_temp_storage = rmm::device_buffer{temp_storage_bytes, stream};

  // Run reduction
  hipcub::DeviceReduce::Reduce(d_temp_storage.data(),
                            temp_storage_bytes,
                            d_in,
                            dev_result.data(),
                            num_items,
                            binary_op,
                            identity,
                            stream);
  return std::move(dev_result);
}

/**
 * @brief stores the minimum and maximum
 * values that have been encountered so far
 *
 */
template <typename T>
struct minmax_pair {
  T min_val;
  T max_val;
  bool min_valid;
  bool max_valid;

  __host__ __device__ minmax_pair()
    : min_val(cudf::DeviceMin::identity<T>()),
      max_val(cudf::DeviceMax::identity<T>()),
      min_valid(false),
      max_valid(false){};
  __host__ __device__ minmax_pair(T val, bool valid_)
    : min_val(val), max_val(val), min_valid(valid_), max_valid(valid_){};
  __host__ __device__ minmax_pair(T min_val_, bool min_valid_, T max_val_, bool max_valid_)
    : min_val(min_val_), max_val(max_val_), min_valid(min_valid_), max_valid(max_valid_){};
};

/**
 * @brief functor that accepts two minmax_pairs and returns a
 * minmax_pair whose minimum and maximum values are the min() and max()
 * respectively of the minimums and maximums of the input pairs. Respects
 * validity.
 *
 */
template <typename T, bool has_nulls = true>
struct minmax_binary_op
  : public thrust::binary_function<minmax_pair<T>, minmax_pair<T>, minmax_pair<T>> {
  __device__ minmax_pair<T> operator()(minmax_pair<T> const &lhs, minmax_pair<T> const &rhs) const
  {
    T const x_min = (lhs.min_valid || !has_nulls) ? lhs.min_val : cudf::DeviceMin::identity<T>();
    T const y_min = (rhs.min_valid || !has_nulls) ? rhs.min_val : cudf::DeviceMin::identity<T>();
    T const x_max = (lhs.max_valid || !has_nulls) ? lhs.max_val : cudf::DeviceMax::identity<T>();
    T const y_max = (rhs.max_valid || !has_nulls) ? rhs.max_val : cudf::DeviceMax::identity<T>();

    // The only invalid situation is if we compare two invalid values.
    // Otherwise, we are certain to select a valid value due to the
    // identity functions above changing the comparison value.
    bool const valid_min_result = !has_nulls || lhs.min_valid || rhs.min_valid;
    bool const valid_max_result = !has_nulls || lhs.max_valid || rhs.max_valid;

    return minmax_pair<T>{
      thrust::min(x_min, y_min), valid_min_result, thrust::max(x_max, y_max), valid_max_result};
  }
};

template <typename T>
struct minmax_iterfunctor {
  column_device_view d_col;
  __device__ minmax_pair<T> operator()(size_type index)
  {
    return minmax_pair<T>(d_col.element<T>(index), d_col.is_valid(index));
  };
};
/**
 * @brief functor that calls thrust::transform_reduce to produce a std::pair
 * of scalars that represent the minimum and maximum values of the input data
 * respectively. Note that dictionaries and non-relationally comparable objects
 * are not supported.
 *
 */
struct minmax_functor {
  template <typename T>
  std::pair<std::unique_ptr<scalar>, std::unique_ptr<scalar>> operator()(
    const cudf::column_view &col, rmm::mr::device_memory_resource *mr, hipStream_t stream)
  {
    auto device_col  = column_device_view::create(col, stream);
    using OutputType = minmax_pair<T>;

    // compute minimum and maximum values
    auto dev_result = [&]() -> rmm::device_scalar<OutputType> {
      auto op = minmax_iterfunctor<T>{*device_col};
      auto begin =
        thrust::make_transform_iterator(thrust::make_counting_iterator<size_type>(0), op);
      // auto end = begin + col.size();

      if (col.nullable()) {
        auto binary_op = minmax_binary_op<T, true>{};
        return reduce_device(begin, col.size(), binary_op, mr, stream);
      } else {
        auto binary_op = minmax_binary_op<T, false>{};
        return reduce_device(begin, col.size(), binary_op, mr, stream);
      }
    }();

    using ScalarType = cudf::scalar_type_t<T>;
    auto min         = new ScalarType(T{}, false, stream, mr);
    auto max         = new ScalarType(T{}, false, stream, mr);

    device_single_thread(
      [result    = dev_result.data(),
       min_data  = min->data(),
       min_valid = min->validity_data(),
       max_data  = max->data(),
       max_valid = max->validity_data()] __device__() mutable {
        *min_data  = result->min_val;
        *min_valid = result->min_valid;
        *max_data  = result->max_val;
        *max_valid = result->max_valid;
      },
      stream);
    return {std::move(std::unique_ptr<scalar>(min)), std::move(std::unique_ptr<scalar>(max))};
  }
};

template <>
std::pair<std::unique_ptr<scalar>, std::unique_ptr<scalar>> minmax_functor::
operator()<cudf::dictionary32>(const cudf::column_view &col,
                               rmm::mr::device_memory_resource *mr,
                               hipStream_t stream)
{
  CUDF_FAIL("dictionary type not supported");
}

template <>
std::pair<std::unique_ptr<scalar>, std::unique_ptr<scalar>> minmax_functor::
operator()<cudf::string_view>(const cudf::column_view &col,
                              rmm::mr::device_memory_resource *mr,
                              hipStream_t stream)
{
  CUDF_FAIL("string type not supported");
}

template <>
std::pair<std::unique_ptr<scalar>, std::unique_ptr<scalar>> minmax_functor::
operator()<cudf::list_view>(const cudf::column_view &col,
                            rmm::mr::device_memory_resource *mr,
                            hipStream_t stream)
{
  CUDF_FAIL("list type not supported");
}

template <>
std::pair<std::unique_ptr<scalar>, std::unique_ptr<scalar>> minmax_functor::
operator()<cudf::struct_view>(const cudf::column_view &col,
                              rmm::mr::device_memory_resource *mr,
                              hipStream_t stream)
{
  CUDF_FAIL("struct type not supported");
}

// unable to support fixed point due to DeviceMin/DeviceMax not supporting fixed point
template <>
std::pair<std::unique_ptr<scalar>, std::unique_ptr<scalar>> minmax_functor::
operator()<numeric::decimal32>(const cudf::column_view &col,
                               rmm::mr::device_memory_resource *mr,
                               hipStream_t stream)
{
  CUDF_FAIL("fixed-point type not supported");
}

template <>
std::pair<std::unique_ptr<scalar>, std::unique_ptr<scalar>> minmax_functor::
operator()<numeric::decimal64>(const cudf::column_view &col,
                               rmm::mr::device_memory_resource *mr,
                               hipStream_t stream)
{
  CUDF_FAIL("fixed-point type not supported");
}

}  // namespace

std::pair<std::unique_ptr<scalar>, std::unique_ptr<scalar>> minmax(
  const cudf::column_view &col,
  rmm::mr::device_memory_resource *mr = rmm::mr::get_current_device_resource(),
  hipStream_t stream                 = 0)
{
  return type_dispatcher(col.type(), minmax_functor{}, col, mr, stream);
}
}  // namespace detail

/**
 * @copydoc cudf::minmax
 */
std::pair<std::unique_ptr<scalar>, std::unique_ptr<scalar>> minmax(
  const cudf::column_view &col, rmm::mr::device_memory_resource *mr)
{
  return cudf::detail::minmax(col, mr);
}

}  // namespace cudf
