#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2020, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */
// The translation unit for reduction `minmax`

#include <thrust/transform_reduce.h>
#include <cudf/column/column_device_view.cuh>
#include <cudf/column/column_view.hpp>
#include <cudf/detail/utilities/device_operators.cuh>
#include <cudf/reduction.hpp>
#include <cudf/scalar/scalar_factories.hpp>
#include <type_traits>

namespace cudf {
namespace detail {

namespace {
/**
 * @brief stores the minimum and maximum
 * values that have been encountered so far
 *
 */
template <typename T>
struct minmax_pair {
  T min_val;
  T max_val;
  bool min_valid;
  bool max_valid;

  __host__ __device__ minmax_pair()
    : min_val(cudf::DeviceMin::identity<T>()),
      max_val(cudf::DeviceMax::identity<T>()),
      min_valid(false),
      max_valid(false){};
  __host__ __device__ minmax_pair(T val, bool valid_)
    : min_val(val), max_val(val), min_valid(valid_), max_valid(valid_){};
  __host__ __device__ minmax_pair(T min_val_, bool min_valid_, T max_val_, bool max_valid_)
    : min_val(min_val_), max_val(max_val_), min_valid(min_valid_), max_valid(max_valid_){};
};

/**
 * @brief functor that accepts two minmax_pairs and returns a
 * minmax_pair whose minimum and maximum values are the min() and max()
 * respectively of the minimums and maximums of the input pairs. Respects
 * validity.
 *
 */
template <typename T, bool has_nulls = true>
struct minmax_binary_op
  : public thrust::binary_function<minmax_pair<T>, minmax_pair<T>, minmax_pair<T>> {
  __device__ minmax_pair<T> operator()(minmax_pair<T> const &lhs, minmax_pair<T> const &rhs) const
  {
    T const x_min = (lhs.min_valid || !has_nulls) ? lhs.min_val : cudf::DeviceMin::identity<T>();
    T const y_min = (rhs.min_valid || !has_nulls) ? rhs.min_val : cudf::DeviceMin::identity<T>();
    T const x_max = (lhs.max_valid || !has_nulls) ? lhs.max_val : cudf::DeviceMax::identity<T>();
    T const y_max = (rhs.max_valid || !has_nulls) ? rhs.max_val : cudf::DeviceMax::identity<T>();

    // The only invalid situation is if we compare two invalid values.
    // Otherwise, we are certain to select a valid value due to the
    // identity functions above changing the comparison value.
    bool const valid_min_result = !has_nulls || lhs.min_valid || rhs.min_valid;
    bool const valid_max_result = !has_nulls || lhs.max_valid || rhs.max_valid;

    return minmax_pair<T>{
      thrust::min(x_min, y_min), valid_min_result, thrust::max(x_max, y_max), valid_max_result};
  }
};

template <typename T>
struct minmax_iterfunctor {
  column_device_view d_col;
  __device__ minmax_pair<T> operator()(size_type index)
  {
    return minmax_pair<T>(d_col.element<T>(index), d_col.is_valid(index));
  };
};
/**
 * @brief functor that calls thrust::transform_reduce to produce a std::pair
 * of scalars that represent the minimum and maximum values of the input data
 * respectively. Note that dictionaries and non-relationally comparable objects
 * are not supported.
 *
 */
struct minmax_functor {
  template <typename T>
  std::pair<std::unique_ptr<scalar>, std::unique_ptr<scalar>> operator()(
    const cudf::column_view &col, rmm::mr::device_memory_resource *mr, hipStream_t stream)
  {
    auto device_col = column_device_view::create(col, stream);

    // compute minimum and maximum values
    minmax_pair<T> const result = [&]() -> minmax_pair<T> {
      auto op = minmax_iterfunctor<T>{*device_col};
      auto begin =
        thrust::make_transform_iterator(thrust::make_counting_iterator<size_type>(0), op);
      auto end = begin + col.size();

      if (col.nullable()) {
        return thrust::reduce(rmm::exec_policy(stream)->on(stream),
                              begin,
                              end,
                              minmax_pair<T>{},
                              minmax_binary_op<T, true>{});
      } else {
        return thrust::reduce(rmm::exec_policy(stream)->on(stream),
                              begin,
                              end,
                              minmax_pair<T>{},
                              minmax_binary_op<T, false>{});
      }
    }();

    std::unique_ptr<scalar> min =
      make_fixed_width_scalar<T>(result.min_val, result.min_valid, stream, mr);
    std::unique_ptr<scalar> max =
      make_fixed_width_scalar<T>(result.max_val, result.max_valid, stream, mr);
    return {std::move(min), std::move(max)};
  }
};

template <>
std::pair<std::unique_ptr<scalar>, std::unique_ptr<scalar>> minmax_functor::
operator()<cudf::dictionary32>(const cudf::column_view &col,
                               rmm::mr::device_memory_resource *mr,
                               hipStream_t stream)
{
  CUDF_FAIL("dictionary type not supported");
}

template <>
std::pair<std::unique_ptr<scalar>, std::unique_ptr<scalar>> minmax_functor::
operator()<cudf::string_view>(const cudf::column_view &col,
                              rmm::mr::device_memory_resource *mr,
                              hipStream_t stream)
{
  CUDF_FAIL("string type not supported");
}

template <>
std::pair<std::unique_ptr<scalar>, std::unique_ptr<scalar>> minmax_functor::
operator()<cudf::list_view>(const cudf::column_view &col,
                            rmm::mr::device_memory_resource *mr,
                            hipStream_t stream)
{
  CUDF_FAIL("list type not supported");
}

template <>
std::pair<std::unique_ptr<scalar>, std::unique_ptr<scalar>> minmax_functor::
operator()<cudf::struct_view>(const cudf::column_view &col,
                              rmm::mr::device_memory_resource *mr,
                              hipStream_t stream)
{
  CUDF_FAIL("struct type not supported");
}

// unable to support fixed point due to DeviceMin/DeviceMax not supporting fixed point
template <>
std::pair<std::unique_ptr<scalar>, std::unique_ptr<scalar>> minmax_functor::
operator()<numeric::decimal32>(const cudf::column_view &col,
                               rmm::mr::device_memory_resource *mr,
                               hipStream_t stream)
{
  CUDF_FAIL("fixed-point type not supported");
}

template <>
std::pair<std::unique_ptr<scalar>, std::unique_ptr<scalar>> minmax_functor::
operator()<numeric::decimal64>(const cudf::column_view &col,
                               rmm::mr::device_memory_resource *mr,
                               hipStream_t stream)
{
  CUDF_FAIL("fixed-point type not supported");
}

}  // namespace

std::pair<std::unique_ptr<scalar>, std::unique_ptr<scalar>> minmax(
  const cudf::column_view &col,
  rmm::mr::device_memory_resource *mr = rmm::mr::get_current_device_resource(),
  hipStream_t stream                 = 0)
{
  return type_dispatcher(col.type(), minmax_functor{}, col, mr, stream);
}
}  // namespace detail

/**
 * @copydoc cudf::minmax
 */
std::pair<std::unique_ptr<scalar>, std::unique_ptr<scalar>> minmax(
  const cudf::column_view &col, rmm::mr::device_memory_resource *mr)
{
  return cudf::detail::minmax(col, mr);
}

}  // namespace cudf
