#include "hip/hip_runtime.h"
#include <cudf/column/column_device_view.cuh>
#include <cudf/column/column_factories.hpp>
#include <cudf/column/column_view.hpp>
#include <cudf/detail/iterator.cuh>
#include <cudf/detail/nvtx/ranges.hpp>

#include <cudf/null_mask.hpp>
#include <cudf/utilities/error.hpp>
#include <cudf/utilities/type_dispatcher.hpp>

#include <cudf/detail/aggregation/aggregation.hpp>
#include <cudf/detail/copy.hpp>
#include <cudf/detail/utilities/device_atomics.cuh>
#include <cudf/null_mask.hpp>
#include <cudf/reduction.hpp>

namespace cudf {
namespace detail {
/**
 * @brief Dispatcher for running Scan operation on input column
 * Dispatches scan operation on `Op` and creates output column
 *
 * @tparam Op device binary operator
 */
template <typename Op>
struct ScanDispatcher {
 private:
  template <typename T>
  static constexpr bool is_string_supported()
  {
    return std::is_same<T, string_view>::value &&
           (std::is_same<Op, cudf::DeviceMin>::value || std::is_same<Op, cudf::DeviceMax>::value);
  }
  // return true if T is arithmetic type (including bool)
  template <typename T>
  static constexpr bool is_supported()
  {
    return std::is_arithmetic<T>::value || is_string_supported<T>();
  }

  // for arithmetic types
  template <typename T, std::enable_if_t<std::is_arithmetic<T>::value, T>* = nullptr>
  auto exclusive_scan(const column_view& input_view,
                      null_policy null_handling,
                      rmm::mr::device_memory_resource* mr,
                      hipStream_t stream)
  {
    const size_type size = input_view.size();
    auto output_column =
      detail::allocate_like(input_view, size, mask_allocation_policy::NEVER, mr, stream);
    if (null_handling == null_policy::EXCLUDE) {
      output_column->set_null_mask(copy_bitmask(input_view, stream, mr), input_view.null_count());
    }
    mutable_column_view output = output_column->mutable_view();
    auto d_input               = column_device_view::create(input_view, stream);

    if (input_view.has_nulls()) {
      auto input = make_null_replacement_iterator(*d_input, Op::template identity<T>());
      thrust::exclusive_scan(rmm::exec_policy(stream)->on(stream),
                             input,
                             input + size,
                             output.data<T>(),
                             Op::template identity<T>(),
                             Op{});
    } else {
      auto input = d_input->begin<T>();
      thrust::exclusive_scan(rmm::exec_policy(stream)->on(stream),
                             input,
                             input + size,
                             output.data<T>(),
                             Op::template identity<T>(),
                             Op{});
    }

    CHECK_CUDA(stream);
    return output_column;
  }

  // for string type
  template <typename T, std::enable_if_t<is_string_supported<T>(), T>* = nullptr>
  std::unique_ptr<column> exclusive_scan(const column_view& input_view,
                                         null_policy null_handling,
                                         rmm::mr::device_memory_resource* mr,
                                         hipStream_t stream)
  {
    CUDF_FAIL("String types supports only inclusive min/max for `cudf::scan`");
  }

  rmm::device_buffer mask_inclusive_scan(const column_view& input_view,
                                         rmm::mr::device_memory_resource* mr,
                                         hipStream_t stream)
  {
    rmm::device_buffer mask =
      create_null_mask(input_view.size(), mask_state::UNINITIALIZED, stream, mr);
    auto d_input = column_device_view::create(input_view, stream);
    auto v       = detail::make_validity_iterator(*d_input);
    auto first_null_position =
      thrust::find_if_not(
        rmm::exec_policy(stream)->on(stream), v, v + input_view.size(), thrust::identity<bool>{}) -
      v;
    cudf::set_null_mask(
      static_cast<cudf::bitmask_type*>(mask.data()), 0, first_null_position, true);
    cudf::set_null_mask(
      static_cast<cudf::bitmask_type*>(mask.data()), first_null_position, input_view.size(), false);
    return mask;
  }

  // for arithmetic types
  template <typename T, std::enable_if_t<std::is_arithmetic<T>::value, T>* = nullptr>
  auto inclusive_scan(const column_view& input_view,
                      null_policy null_handling,
                      rmm::mr::device_memory_resource* mr,
                      hipStream_t stream)
  {
    const size_type size = input_view.size();
    auto output_column =
      detail::allocate_like(input_view, size, mask_allocation_policy::NEVER, mr, stream);
    if (null_handling == null_policy::EXCLUDE) {
      output_column->set_null_mask(copy_bitmask(input_view, stream, mr), input_view.null_count());
    } else {
      if (input_view.nullable()) {
        output_column->set_null_mask(mask_inclusive_scan(input_view, mr, stream),
                                     cudf::UNKNOWN_NULL_COUNT);
      }
    }

    auto d_input               = column_device_view::create(input_view, stream);
    mutable_column_view output = output_column->mutable_view();

    if (input_view.has_nulls()) {
      auto input = make_null_replacement_iterator(*d_input, Op::template identity<T>());
      thrust::inclusive_scan(
        rmm::exec_policy(stream)->on(stream), input, input + size, output.data<T>(), Op{});
    } else {
      auto input = d_input->begin<T>();
      thrust::inclusive_scan(
        rmm::exec_policy(stream)->on(stream), input, input + size, output.data<T>(), Op{});
    }

    CHECK_CUDA(stream);
    return output_column;
  }

  // for string type
  template <typename T, std::enable_if_t<is_string_supported<T>(), T>* = nullptr>
  std::unique_ptr<column> inclusive_scan(const column_view& input_view,
                                         null_policy null_handling,
                                         rmm::mr::device_memory_resource* mr,
                                         hipStream_t stream)
  {
    const size_type size = input_view.size();
    rmm::device_vector<T> result(size);

    auto d_input = column_device_view::create(input_view, stream);

    if (input_view.has_nulls()) {
      auto input = make_null_replacement_iterator(*d_input, Op::template identity<T>());
      thrust::inclusive_scan(
        rmm::exec_policy(stream)->on(stream), input, input + size, result.data().get(), Op{});
    } else {
      auto input = d_input->begin<T>();
      thrust::inclusive_scan(
        rmm::exec_policy(stream)->on(stream), input, input + size, result.data().get(), Op{});
    }
    CHECK_CUDA(stream);

    auto output_column = make_strings_column(result, Op::template identity<T>(), stream, mr);
    if (null_handling == null_policy::EXCLUDE) {
      output_column->set_null_mask(copy_bitmask(input_view, stream, mr), input_view.null_count());
    } else {
      if (input_view.nullable()) {
        output_column->set_null_mask(mask_inclusive_scan(input_view, mr, stream),
                                     cudf::UNKNOWN_NULL_COUNT);
      }
    }
    return output_column;
  }

 public:
  /**
   * @brief creates new column from input column by applying scan operation
   *
   * @param input     input column view
   * @param inclusive inclusive or exclusive scan
   * @param mr Device memory resource used to allocate the returned column's device memory
   * @param stream CUDA stream used for device memory operations and kernel launches.
   * @return
   *
   * @tparam T type of input column
   */
  template <typename T, typename std::enable_if_t<is_supported<T>(), T>* = nullptr>
  std::unique_ptr<column> operator()(const column_view& input,
                                     scan_type inclusive,
                                     null_policy null_handling,
                                     rmm::mr::device_memory_resource* mr,
                                     hipStream_t stream)
  {
    std::unique_ptr<column> output;
    if (inclusive == scan_type::INCLUSIVE)
      output = inclusive_scan<T>(input, null_handling, mr, stream);
    else
      output = exclusive_scan<T>(input, null_handling, mr, stream);
    if (null_handling == null_policy::EXCLUDE) {
      CUDF_EXPECTS(input.null_count() == output->null_count(),
                   "Input / output column null count mismatch");
    }
    return output;
  }

  template <typename T, typename std::enable_if_t<!is_supported<T>(), T>* = nullptr>
  std::unique_ptr<column> operator()(const column_view& input,
                                     scan_type inclusive,
                                     null_policy null_handling,
                                     rmm::mr::device_memory_resource* mr,
                                     hipStream_t stream)
  {
    CUDF_FAIL("Non-arithmetic types not supported for `cudf::scan`");
  }
};

std::unique_ptr<column> scan(
  const column_view& input,
  std::unique_ptr<aggregation> const& agg,
  scan_type inclusive,
  null_policy null_handling,
  rmm::mr::device_memory_resource* mr = rmm::mr::get_current_device_resource(),
  hipStream_t stream                 = 0)
{
  CUDF_EXPECTS(is_numeric(input.type()) || is_compound(input.type()),
               "Unexpected non-numeric or non-string type.");

  switch (agg->kind) {
    case aggregation::SUM:
      return cudf::type_dispatcher(input.type(),
                                   ScanDispatcher<cudf::DeviceSum>(),
                                   input,
                                   inclusive,
                                   null_handling,
                                   mr,
                                   stream);
    case aggregation::MIN:
      return cudf::type_dispatcher(input.type(),
                                   ScanDispatcher<cudf::DeviceMin>(),
                                   input,
                                   inclusive,
                                   null_handling,
                                   mr,
                                   stream);
    case aggregation::MAX:
      return cudf::type_dispatcher(input.type(),
                                   ScanDispatcher<cudf::DeviceMax>(),
                                   input,
                                   inclusive,
                                   null_handling,
                                   mr,
                                   stream);
    case aggregation::PRODUCT:
      return cudf::type_dispatcher(input.type(),
                                   ScanDispatcher<cudf::DeviceProduct>(),
                                   input,
                                   inclusive,
                                   null_handling,
                                   mr,
                                   stream);
    default: CUDF_FAIL("Unsupported aggregation operator for scan");
  }
}
}  // namespace detail

std::unique_ptr<column> scan(const column_view& input,
                             std::unique_ptr<aggregation> const& agg,
                             scan_type inclusive,
                             null_policy null_handling,
                             rmm::mr::device_memory_resource* mr)
{
  CUDF_FUNC_RANGE();
  return detail::scan(input, agg, inclusive, null_handling, mr);
}

}  // namespace cudf
