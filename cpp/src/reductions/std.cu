/*
 * Copyright (c) 2019, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */
// The translation unit for reduction `standard deviation`

#include "reduction_functions.cuh"
#include "compound.cuh"


// @param[in] ddof Delta Degrees of Freedom used for `std`, `var`.
//                 The divisor used in calculations is N - ddof, where N represents the number of elements.

gdf_scalar cudf::reduction::standard_deviation(gdf_column const& col, gdf_dtype const output_dtype, gdf_size_type ddof, hipStream_t stream)
{
    // TODO: add cuda version check when the fix is available
#if !defined(__CUDACC_DEBUG__)
    using reducer = cudf::reduction::compound::element_type_dispatcher<cudf::reduction::op::standard_deviation>;
    return cudf::type_dispatcher(col.dtype, reducer(), col, output_dtype, ddof, stream);
#else
    // workaround for bug 200529165 which causes compilation error only at device debug build
    // the bug will be fixed at cuda 10.2
    CUDF_FAIL("var/std reductions are not supported at debug build.");
#endif
}


