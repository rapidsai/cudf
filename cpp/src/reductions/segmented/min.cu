#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2022-2023, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "simple.cuh"

#include <cudf/reduction/detail/reduction_functions.hpp>

namespace cudf {
namespace reduction {
namespace detail {

std::unique_ptr<cudf::column> segmented_min(
  column_view const& col,
  device_span<size_type const> offsets,
  data_type const output_dtype,
  null_policy null_handling,
  std::optional<std::reference_wrapper<scalar const>> init,
  rmm::cuda_stream_view stream,
  rmm::mr::device_memory_resource* mr)
{
  CUDF_EXPECTS(col.type() == output_dtype,
               "segmented_min() operation requires matching output type");
  using reducer = simple::detail::same_column_type_dispatcher<op::min>;
  return cudf::type_dispatcher(
    col.type(), reducer{}, col, offsets, null_handling, init, stream, mr);
}
}  // namespace detail
}  // namespace reduction
}  // namespace cudf
