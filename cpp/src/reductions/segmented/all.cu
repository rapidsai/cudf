#include "hip/hip_runtime.h"
/*
 * SPDX-FileCopyrightText: Copyright (c) 2022-2025, NVIDIA CORPORATION.
 * SPDX-License-Identifier: Apache-2.0
 */

#include "simple.cuh"

#include <cudf/reduction/detail/segmented_reduction_functions.hpp>
#include <cudf/utilities/memory_resource.hpp>

namespace cudf {
namespace reduction {
namespace detail {

std::unique_ptr<cudf::column> segmented_all(
  column_view const& col,
  device_span<size_type const> offsets,
  cudf::data_type const output_dtype,
  null_policy null_handling,
  std::optional<std::reference_wrapper<scalar const>> init,
  rmm::cuda_stream_view stream,
  rmm::device_async_resource_ref mr)
{
  CUDF_EXPECTS(output_dtype == cudf::data_type(cudf::type_id::BOOL8),
               "segmented_all() operation requires output type `BOOL8`");

  using reducer = simple::detail::bool_result_column_dispatcher<op::min>;
  // A minimum over bool types is used to implement all()
  return cudf::type_dispatcher(
    col.type(), reducer{}, col, offsets, null_handling, init, stream, mr);
}

}  // namespace detail
}  // namespace reduction
}  // namespace cudf
