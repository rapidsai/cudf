#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2022-2024, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "simple.cuh"

#include <cudf/reduction/detail/reduction_functions.hpp>

#include <rmm/resource_ref.hpp>

namespace cudf {
namespace reduction {
namespace detail {

std::unique_ptr<cudf::column> segmented_max(
  column_view const& col,
  device_span<size_type const> offsets,
  cudf::data_type const output_dtype,
  null_policy null_handling,
  std::optional<std::reference_wrapper<scalar const>> init,
  rmm::cuda_stream_view stream,
  rmm::device_async_resource_ref mr)
{
  CUDF_EXPECTS(col.type() == output_dtype,
               "segmented_max() operation requires matching output type");
  using reducer = simple::detail::same_column_type_dispatcher<op::max>;
  return cudf::type_dispatcher(
    col.type(), reducer{}, col, offsets, null_handling, init, stream, mr);
}
}  // namespace detail
}  // namespace reduction
}  // namespace cudf
