#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019-2020, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <cudf/detail/reduction_functions.hpp>
#include <cudf/dictionary/dictionary_column_view.hpp>
#include <reductions/simple_segmented.cuh>

#include <rmm/cuda_stream_view.hpp>

namespace cudf {
namespace reduction {

std::unique_ptr<cudf::column> segmented_max(column_view const& col,
                                            column_view const& offsets,
                                            cudf::data_type const output_dtype,
                                            rmm::cuda_stream_view stream,
                                            rmm::mr::device_memory_resource* mr)
{
  CUDF_EXPECTS(col.type() == output_dtype, "max() operation requires matching output type");
  return cudf::type_dispatcher(col.type(),
                               simple::same_column_type_dispatcher<cudf::reduction::op::max>{},
                               col,
                               offsets,
                               stream,
                               mr);
}

}  // namespace reduction
}  // namespace cudf
