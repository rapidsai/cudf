#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2021-2023, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <reductions/scan/scan.cuh>

#include <cudf/column/column_device_view.cuh>
#include <cudf/column/column_factories.hpp>
#include <cudf/detail/copy.hpp>
#include <cudf/detail/gather.hpp>
#include <cudf/detail/iterator.cuh>
#include <cudf/detail/null_mask.hpp>
#include <cudf/detail/structs/utilities.hpp>
#include <cudf/reduction.hpp>
#include <cudf/strings/detail/scan.hpp>
#include <cudf/structs/detail/scan.hpp>

#include <rmm/cuda_stream_view.hpp>
#include <rmm/exec_policy.hpp>

#include <thrust/find.h>
#include <thrust/functional.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/scan.h>

#include <type_traits>

namespace cudf {
namespace detail {

// logical-and scan of the null mask of the input view
std::pair<rmm::device_buffer, size_type> mask_scan(column_view const& input_view,
                                                   scan_type inclusive,
                                                   rmm::cuda_stream_view stream,
                                                   rmm::mr::device_memory_resource* mr)
{
  rmm::device_buffer mask =
    detail::create_null_mask(input_view.size(), mask_state::UNINITIALIZED, stream, mr);
  auto d_input   = column_device_view::create(input_view, stream);
  auto valid_itr = detail::make_validity_iterator(*d_input);

  auto first_null_position = [&] {
    size_type const first_null =
      thrust::find_if_not(
        rmm::exec_policy(stream), valid_itr, valid_itr + input_view.size(), thrust::identity{}) -
      valid_itr;
    size_type const exclusive_offset = (inclusive == scan_type::EXCLUSIVE) ? 1 : 0;
    return std::min(input_view.size(), first_null + exclusive_offset);
  }();

  set_null_mask(static_cast<bitmask_type*>(mask.data()), 0, first_null_position, true, stream);
  set_null_mask(
    static_cast<bitmask_type*>(mask.data()), first_null_position, input_view.size(), false, stream);
  return {std::move(mask), input_view.size() - first_null_position};
}

namespace {

template <typename Op, typename T>
struct scan_functor {
  static std::unique_ptr<column> invoke(column_view const& input_view,
                                        bitmask_type const*,
                                        rmm::cuda_stream_view stream,
                                        rmm::mr::device_memory_resource* mr)
  {
    auto output_column = detail::allocate_like(
      input_view, input_view.size(), mask_allocation_policy::NEVER, stream, mr);
    mutable_column_view result = output_column->mutable_view();

    auto d_input = column_device_view::create(input_view, stream);
    auto const begin =
      make_null_replacement_iterator(*d_input, Op::template identity<T>(), input_view.has_nulls());
    thrust::inclusive_scan(
      rmm::exec_policy(stream), begin, begin + input_view.size(), result.data<T>(), Op{});

    CUDF_CHECK_CUDA(stream.value());
    return output_column;
  }
};

template <typename Op>
struct scan_functor<Op, cudf::string_view> {
  static std::unique_ptr<column> invoke(column_view const& input_view,
                                        bitmask_type const* mask,
                                        rmm::cuda_stream_view stream,
                                        rmm::mr::device_memory_resource* mr)
  {
    return cudf::strings::detail::scan_inclusive<Op>(input_view, mask, stream, mr);
  }
};

template <typename Op>
struct scan_functor<Op, cudf::struct_view> {
  static std::unique_ptr<column> invoke(column_view const& input,
                                        bitmask_type const*,
                                        rmm::cuda_stream_view stream,
                                        rmm::mr::device_memory_resource* mr)
  {
    return cudf::structs::detail::scan_inclusive<Op>(input, stream, mr);
  }
};

/**
 * @brief Dispatcher for running a Scan operation on an input column
 *
 * @tparam Op device binary operator
 */
template <typename Op>
struct scan_dispatcher {
 private:
  template <typename T>
  static constexpr bool is_supported()
  {
    if constexpr (std::is_same_v<T, cudf::struct_view>) {
      return std::is_same_v<Op, DeviceMin> || std::is_same_v<Op, DeviceMax>;
    } else {
      return std::is_invocable_v<Op, T, T> && !cudf::is_dictionary<T>();
    }
  }

 public:
  /**
   * @brief Creates a new column from the input column by applying the scan operation
   *
   * @param input Input column view
   * @param null_handling How null row entries are to be processed
   * @param stream CUDA stream used for device memory operations and kernel launches.
   * @param mr Device memory resource used to allocate the returned column's device memory
   * @return
   *
   * @tparam T type of input column
   */
  template <typename T, std::enable_if_t<is_supported<T>()>* = nullptr>
  std::unique_ptr<column> operator()(column_view const& input,
                                     bitmask_type const* output_mask,
                                     rmm::cuda_stream_view stream,
                                     rmm::mr::device_memory_resource* mr)
  {
    return scan_functor<Op, T>::invoke(input, output_mask, stream, mr);
  }

  template <typename T, typename... Args>
  std::enable_if_t<!is_supported<T>(), std::unique_ptr<column>> operator()(Args&&...)
  {
    CUDF_FAIL("Unsupported type for inclusive scan operation");
  }
};

}  // namespace

std::unique_ptr<column> scan_inclusive(column_view const& input,
                                       scan_aggregation const& agg,
                                       null_policy null_handling,
                                       rmm::cuda_stream_view stream,
                                       rmm::mr::device_memory_resource* mr)
{
  auto [mask, null_count] = [&] {
    if (null_handling == null_policy::EXCLUDE) {
      return std::make_pair(std::move(detail::copy_bitmask(input, stream, mr)), input.null_count());
    } else if (input.nullable()) {
      return mask_scan(input, scan_type::INCLUSIVE, stream, mr);
    }
    return std::make_pair(rmm::device_buffer{}, size_type{0});
  }();

  auto output = scan_agg_dispatch<scan_dispatcher>(
    input, agg, static_cast<bitmask_type*>(mask.data()), stream, mr);
  output->set_null_mask(mask, null_count);

  // If the input is a structs column, we also need to push down nulls from the parent output column
  // into the children columns.
  if (input.type().id() == type_id::STRUCT && output->has_nulls()) {
    auto const num_rows   = output->size();
    auto const null_count = output->null_count();
    auto content          = output->release();

    // Build new children columns.
    auto const null_mask = reinterpret_cast<bitmask_type const*>(content.null_mask->data());
    std::for_each(content.children.begin(),
                  content.children.end(),
                  [null_mask, null_count, stream, mr](auto& child) {
                    child = structs::detail::superimpose_nulls(
                      null_mask, null_count, std::move(child), stream, mr);
                  });

    // Replace the children columns.
    output = cudf::make_structs_column(
      num_rows, std::move(content.children), null_count, std::move(*content.null_mask), stream, mr);
  }

  return output;
}
}  // namespace detail
}  // namespace cudf
