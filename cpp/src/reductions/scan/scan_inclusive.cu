#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2021, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "scan.cuh"

#include <cudf/column/column_device_view.cuh>
#include <cudf/column/column_factories.hpp>
#include <cudf/detail/copy.hpp>
#include <cudf/detail/iterator.cuh>
#include <cudf/detail/null_mask.hpp>
#include <cudf/null_mask.hpp>
#include <cudf/reduction.hpp>
#include <cudf/strings/detail/gather.cuh>
#include <cudf/table/row_operators.cuh>

#include <rmm/cuda_stream_view.hpp>
#include <rmm/device_uvector.hpp>
#include <rmm/exec_policy.hpp>

#include <thrust/logical.h>
#include <thrust/scan.h>

namespace cudf {
namespace detail {

// logical-and scan of the null mask of the input view
rmm::device_buffer mask_scan(column_view const& input_view,
                             scan_type inclusive,
                             rmm::cuda_stream_view stream,
                             rmm::mr::device_memory_resource* mr)
{
  rmm::device_buffer mask =
    detail::create_null_mask(input_view.size(), mask_state::UNINITIALIZED, stream, mr);
  auto d_input   = column_device_view::create(input_view, stream);
  auto valid_itr = detail::make_validity_iterator(*d_input);

  auto first_null_position = [&] {
    size_type const first_null = thrust::find_if_not(rmm::exec_policy(stream),
                                                     valid_itr,
                                                     valid_itr + input_view.size(),
                                                     thrust::identity<bool>{}) -
                                 valid_itr;
    size_type const exclusive_offset = (inclusive == scan_type::EXCLUSIVE) ? 1 : 0;
    return std::min(input_view.size(), first_null + exclusive_offset);
  }();

  set_null_mask(static_cast<bitmask_type*>(mask.data()), 0, first_null_position, true, stream);
  set_null_mask(
    static_cast<bitmask_type*>(mask.data()), first_null_position, input_view.size(), false, stream);
  return mask;
}

namespace {

/**
 * @brief Strings inclusive scan operator
 *
 * This was specifically created to workaround a thrust issue
 * https://github.com/NVIDIA/thrust/issues/1479
 * where invalid values are passed to the operator.
 *
 * This operator will accept index values, check them and then
 * run the `Op` operation on the individual string_view objects.
 * The returned result is the appropriate index value.
 */
template <typename Op>
struct string_scan_operator {
  column_device_view const col;          ///< strings column device view
  string_view const null_replacement{};  ///< value used when element is null
  bool const has_nulls;                  ///< true if col has null elements

  string_scan_operator(column_device_view const& col, bool has_nulls = true)
    : col{col}, null_replacement{Op::template identity<string_view>()}, has_nulls{has_nulls}
  {
    CUDF_EXPECTS(type_id::STRING == col.type().id(), "the data type mismatch");
    // verify validity bitmask is non-null, otherwise, is_null_nocheck() will crash
    if (has_nulls) CUDF_EXPECTS(col.nullable(), "column with nulls must have a validity bitmask");
  }

  CUDA_DEVICE_CALLABLE
  size_type operator()(size_type lhs, size_type rhs) const
  {
    // thrust::inclusive_scan may pass us garbage values so we need to protect ourselves;
    // in these cases the return value does not matter since the result is not used
    if (lhs < 0 || rhs < 0 || lhs >= col.size() || rhs >= col.size()) return 0;
    string_view d_lhs =
      has_nulls && col.is_null_nocheck(lhs) ? null_replacement : col.element<string_view>(lhs);
    string_view d_rhs =
      has_nulls && col.is_null_nocheck(rhs) ? null_replacement : col.element<string_view>(rhs);
    return Op{}(d_lhs, d_rhs) == d_lhs ? lhs : rhs;
  }
};

/**
 * @brief Dispatcher for running a Scan operation on an input column
 *
 * @tparam Op device binary operator
 */
template <typename Op>
struct scan_dispatcher {
 private:
  template <typename T>
  static constexpr bool is_string_supported()
  {
    return std::is_same<T, string_view>::value &&
           (std::is_same<Op, DeviceMin>::value || std::is_same<Op, DeviceMax>::value);
  }

  template <typename T>
  static constexpr bool is_supported()
  {
    return std::is_arithmetic<T>::value || is_string_supported<T>();
  }

  // for arithmetic types
  template <typename T, std::enable_if_t<std::is_arithmetic<T>::value>* = nullptr>
  auto inclusive_scan(column_view const& input_view,
                      null_policy,
                      rmm::cuda_stream_view stream,
                      rmm::mr::device_memory_resource* mr)
  {
    auto output_column = detail::allocate_like(
      input_view, input_view.size(), mask_allocation_policy::NEVER, stream, mr);
    mutable_column_view result = output_column->mutable_view();

    auto d_input = column_device_view::create(input_view, stream);
    auto const begin =
      make_null_replacement_iterator(*d_input, Op::template identity<T>(), input_view.has_nulls());
    thrust::inclusive_scan(
      rmm::exec_policy(stream), begin, begin + input_view.size(), result.data<T>(), Op{});

    CHECK_CUDA(stream.value());
    return output_column;
  }

  // for string type: only MIN and MAX are supported
  template <typename T, std::enable_if_t<is_string_supported<T>()>* = nullptr>
  std::unique_ptr<column> inclusive_scan(column_view const& input_view,
                                         null_policy,
                                         rmm::cuda_stream_view stream,
                                         rmm::mr::device_memory_resource* mr)
  {
    auto d_input = column_device_view::create(input_view, stream);

    // build indices of the scan operation results
    rmm::device_uvector<size_type> result(input_view.size(), stream);
    thrust::inclusive_scan(rmm::exec_policy(stream),
                           thrust::counting_iterator<size_type>(0),
                           thrust::counting_iterator<size_type>(input_view.size()),
                           result.begin(),
                           string_scan_operator<Op>{*d_input, input_view.has_nulls()});

    // call gather using the indices to build the output column
    return cudf::strings::detail::gather(
      strings_column_view(input_view), result.begin(), result.end(), false, stream, mr);
  }

 public:
  /**
   * @brief Creates a new column from the input column by applying the scan operation
   *
   * @param input Input column view
   * @param null_handling How null row entries are to be processed
   * @param stream CUDA stream used for device memory operations and kernel launches.
   * @param mr Device memory resource used to allocate the returned column's device memory
   * @return
   *
   * @tparam T type of input column
   */
  template <typename T, typename std::enable_if_t<is_supported<T>()>* = nullptr>
  std::unique_ptr<column> operator()(column_view const& input,
                                     null_policy null_handling,
                                     rmm::cuda_stream_view stream,
                                     rmm::mr::device_memory_resource* mr)
  {
    return inclusive_scan<T>(input, null_handling, stream, mr);
  }

  template <typename T, typename... Args>
  std::enable_if_t<!is_supported<T>(), std::unique_ptr<column>> operator()(Args&&...)
  {
    CUDF_FAIL("Non-arithmetic types not supported for inclusive scan");
  }
};

template <bool has_nested_nulls>
std::unique_ptr<column> generate_dense_ranks(column_view const& order_by,
                                             rmm::cuda_stream_view stream,
                                             rmm::mr::device_memory_resource* mr)
{
  auto const flat_order =
    order_by.type().id() == type_id::STRUCT
      ? table_view{std::vector<column_view>{order_by.child_begin(), order_by.child_end()}}
      : table_view{{order_by}};
  auto const d_flat_order = table_device_view::create(flat_order, stream);
  row_equality_comparator<has_nested_nulls> comparator(*d_flat_order, *d_flat_order, true);
  auto ranks = make_fixed_width_column(
    data_type{type_to_id<size_type>()}, order_by.size(), mask_state::UNALLOCATED, stream, mr);
  auto mutable_ranks = ranks->mutable_view();

  if (order_by.type().id() == type_id::STRUCT && order_by.has_nulls()) {
    auto const d_col_order = column_device_view::create(order_by, stream);
    thrust::tabulate(rmm::exec_policy(stream),
                     mutable_ranks.begin<size_type>(),
                     mutable_ranks.end<size_type>(),
                     [comparator, d_col_order = *d_col_order] __device__(size_type row_index) {
                       if (row_index == 0) { return true; }
                       bool const lhs_is_null{d_col_order.is_null(row_index)};
                       bool const rhs_is_null{d_col_order.is_null(row_index - 1)};
                       if (lhs_is_null && rhs_is_null) {
                         return false;
                       } else if (lhs_is_null != rhs_is_null) {
                         return true;
                       }
                       return !comparator(row_index, row_index - 1);
                     });
  } else {
    thrust::tabulate(rmm::exec_policy(stream),
                     mutable_ranks.begin<size_type>(),
                     mutable_ranks.end<size_type>(),
                     [comparator] __device__(size_type row_index) {
                       return row_index == 0 || !comparator(row_index, row_index - 1);
                     });
  }

  thrust::inclusive_scan(rmm::exec_policy(stream),
                         mutable_ranks.begin<size_type>(),
                         mutable_ranks.end<size_type>(),
                         mutable_ranks.begin<size_type>());
  return ranks;
}

template <bool has_nested_nulls>
std::unique_ptr<column> generate_ranks(column_view const& order_by,
                                       rmm::cuda_stream_view stream,
                                       rmm::mr::device_memory_resource* mr)
{
  auto const flat_order =
    order_by.type().id() == type_id::STRUCT
      ? table_view{std::vector<column_view>{order_by.child_begin(), order_by.child_end()}}
      : table_view{{order_by}};
  auto const d_flat_order = table_device_view::create(flat_order, stream);
  row_equality_comparator<has_nested_nulls> comparator(*d_flat_order, *d_flat_order, true);
  auto ranks = make_fixed_width_column(
    data_type{type_to_id<size_type>()}, order_by.size(), mask_state::UNALLOCATED, stream, mr);
  auto mutable_ranks = ranks->mutable_view();

  if (order_by.type().id() == type_id::STRUCT && order_by.has_nulls()) {
    auto const d_col_order = column_device_view::create(order_by, stream);
    thrust::tabulate(rmm::exec_policy(stream),
                     mutable_ranks.begin<size_type>(),
                     mutable_ranks.end<size_type>(),
                     [comparator, d_col_order = *d_col_order] __device__(size_type row_index) {
                       if (row_index == 0) { return 1; }
                       bool const lhs_is_null{d_col_order.is_null(row_index)};
                       bool const rhs_is_null{d_col_order.is_null(row_index - 1)};
                       if (lhs_is_null and rhs_is_null) {
                         return 0;
                       } else if (lhs_is_null != rhs_is_null) {
                         return row_index + 1;
                       }
                       return comparator(row_index, row_index - 1) ? 0 : row_index + 1;
                     });
  } else {
    thrust::tabulate(
      rmm::exec_policy(stream),
      mutable_ranks.begin<size_type>(),
      mutable_ranks.end<size_type>(),
      [comparator] __device__(size_type row_index) {
        return row_index != 0 && comparator(row_index, row_index - 1) ? 0 : row_index + 1;
      });
  }

  thrust::inclusive_scan(rmm::exec_policy(stream),
                         mutable_ranks.begin<size_type>(),
                         mutable_ranks.end<size_type>(),
                         mutable_ranks.begin<size_type>(),
                         DeviceMax{});
  return ranks;
}

}  // namespace

std::unique_ptr<column> inclusive_dense_rank_scan(column_view const& order_by,
                                                  rmm::cuda_stream_view stream,
                                                  rmm::mr::device_memory_resource* mr)
{
  CUDF_EXPECTS(order_by.type().id() != type_id::LIST, "Unsupported list type in dense_rank scan.");
  CUDF_EXPECTS(std::none_of(order_by.child_begin(),
                            order_by.child_end(),
                            [](auto const& col) { return is_nested(col.type()); }),
               "Unsupported nested columns in dense_rank scan.");
  if ((order_by.type().id() == type_id::STRUCT &&
       has_nested_nulls(
         table_view{std::vector<column_view>{order_by.child_begin(), order_by.child_end()}})) ||
      (order_by.type().id() != type_id::STRUCT && order_by.has_nulls())) {
    return generate_dense_ranks<true>(order_by, stream, mr);
  }
  return generate_dense_ranks<false>(order_by, stream, mr);
}

std::unique_ptr<column> inclusive_rank_scan(column_view const& order_by,
                                            rmm::cuda_stream_view stream,
                                            rmm::mr::device_memory_resource* mr)
{
  CUDF_EXPECTS(order_by.type().id() != type_id::LIST, "Unsupported list type in rank scan.");
  CUDF_EXPECTS(std::none_of(order_by.child_begin(),
                            order_by.child_end(),
                            [](auto const& col) { return is_nested(col.type()); }),
               "Unsupported nested columns in rank scan.");
  if ((order_by.type().id() == type_id::STRUCT &&
       has_nested_nulls(
         table_view{std::vector<column_view>{order_by.child_begin(), order_by.child_end()}})) ||
      (order_by.type().id() != type_id::STRUCT && order_by.has_nulls())) {
    return generate_ranks<true>(order_by, stream, mr);
  }
  return generate_ranks<false>(order_by, stream, mr);
}

std::unique_ptr<column> scan_inclusive(
  column_view const& input,
  std::unique_ptr<aggregation> const& agg,
  null_policy null_handling,
  rmm::cuda_stream_view stream,
  rmm::mr::device_memory_resource* mr = rmm::mr::get_current_device_resource())
{
  auto output = scan_agg_dispatch<scan_dispatcher>(input, agg, null_handling, stream, mr);

  if (agg->kind == aggregation::RANK || agg->kind == aggregation::DENSE_RANK) {
    return output;
  } else if (null_handling == null_policy::EXCLUDE) {
    output->set_null_mask(detail::copy_bitmask(input, stream, mr), input.null_count());
  } else if (input.nullable()) {
    output->set_null_mask(mask_scan(input, scan_type::INCLUSIVE, stream, mr), UNKNOWN_NULL_COUNT);
  }

  return output;
}
}  // namespace detail
}  // namespace cudf
