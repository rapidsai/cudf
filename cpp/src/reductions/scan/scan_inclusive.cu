#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2021, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "scan.cuh"

#include <cudf/column/column_device_view.cuh>
#include <cudf/column/column_factories.hpp>
#include <cudf/detail/copy.hpp>
#include <cudf/detail/iterator.cuh>
#include <cudf/detail/null_mask.hpp>
#include <cudf/null_mask.hpp>
#include <cudf/reduction.hpp>
#include <cudf/strings/detail/gather.cuh>

#include <rmm/cuda_stream_view.hpp>
#include <rmm/device_uvector.hpp>
#include <rmm/exec_policy.hpp>

#include <thrust/scan.h>

namespace cudf {
namespace detail {

// logical-and scan of the null mask of the input view
rmm::device_buffer mask_scan(const column_view& input_view,
                             cudf::scan_type inclusive,
                             rmm::cuda_stream_view stream,
                             rmm::mr::device_memory_resource* mr)
{
  rmm::device_buffer mask =
    detail::create_null_mask(input_view.size(), mask_state::UNINITIALIZED, stream, mr);
  auto d_input   = column_device_view::create(input_view, stream);
  auto valid_itr = detail::make_validity_iterator(*d_input);

  auto first_null_position = [&] {
    size_type const first_null = thrust::find_if_not(rmm::exec_policy(stream),
                                                     valid_itr,
                                                     valid_itr + input_view.size(),
                                                     thrust::identity<bool>{}) -
                                 valid_itr;
    size_type const exclusive_offset = (inclusive == scan_type::EXCLUSIVE) ? 1 : 0;
    return std::min(input_view.size(), first_null + exclusive_offset);
  }();

  cudf::detail::set_null_mask(
    static_cast<cudf::bitmask_type*>(mask.data()), 0, first_null_position, true, stream);
  cudf::detail::set_null_mask(static_cast<cudf::bitmask_type*>(mask.data()),
                              first_null_position,
                              input_view.size(),
                              false,
                              stream);
  return mask;
}

namespace {

/**
 * @brief Strings inclusive scan operator
 *
 * This was specifically created to workaround a thrust issue
 * https://github.com/NVIDIA/thrust/issues/1479
 * where invalid values are passed to the operator.
 *
 * This operator will accept index values, check them and then
 * run the `Op` operation on the individual string_view objects.
 * The returned result is the appropriate index value.
 */
template <typename Op>
struct string_scan_operator {
  column_device_view const col;          ///< strings column device view
  string_view const null_replacement{};  ///< value used when element is null
  bool const has_nulls;                  ///< true if col has null elements

  string_scan_operator(column_device_view const& col, bool has_nulls = true)
    : col{col}, null_replacement{Op::template identity<string_view>()}, has_nulls{has_nulls}
  {
    CUDF_EXPECTS(type_id::STRING == col.type().id(), "the data type mismatch");
    // verify validity bitmask is non-null, otherwise, is_null_nocheck() will crash
    if (has_nulls) CUDF_EXPECTS(col.nullable(), "column with nulls must have a validity bitmask");
  }

  CUDA_DEVICE_CALLABLE
  size_type operator()(size_type lhs, size_type rhs) const
  {
    // thrust::inclusive_scan may pass us garbage values so we need to protect ourselves;
    // in these cases the return value does not matter since the result is not used
    if (lhs < 0 || rhs < 0 || lhs >= col.size() || rhs >= col.size()) return 0;
    string_view d_lhs =
      has_nulls && col.is_null_nocheck(lhs) ? null_replacement : col.element<string_view>(lhs);
    string_view d_rhs =
      has_nulls && col.is_null_nocheck(rhs) ? null_replacement : col.element<string_view>(rhs);
    return Op{}(d_lhs, d_rhs) == d_lhs ? lhs : rhs;
  }
};

/**
 * @brief Dispatcher for running a Scan operation on an input column
 *
 * @tparam Op device binary operator
 */
template <typename Op>
struct scan_dispatcher {
 private:
  template <typename T>
  static constexpr bool is_string_supported()
  {
    return std::is_same<T, string_view>::value &&
           (std::is_same<Op, cudf::DeviceMin>::value || std::is_same<Op, cudf::DeviceMax>::value);
  }

  template <typename T>
  static constexpr bool is_supported()
  {
    return std::is_arithmetic<T>::value || is_string_supported<T>();
  }

  // for arithmetic types
  template <typename T, std::enable_if_t<std::is_arithmetic<T>::value>* = nullptr>
  auto inclusive_scan(const column_view& input_view,
                      null_policy,
                      rmm::cuda_stream_view stream,
                      rmm::mr::device_memory_resource* mr)
  {
    auto output_column = detail::allocate_like(
      input_view, input_view.size(), mask_allocation_policy::NEVER, stream, mr);
    mutable_column_view result = output_column->mutable_view();

    auto d_input = column_device_view::create(input_view, stream);
    auto const begin =
      make_null_replacement_iterator(*d_input, Op::template identity<T>(), input_view.has_nulls());
    thrust::inclusive_scan(
      rmm::exec_policy(stream), begin, begin + input_view.size(), result.data<T>(), Op{});

    CHECK_CUDA(stream.value());
    return output_column;
  }

  // for string type: only MIN and MAX are supported
  template <typename T, std::enable_if_t<is_string_supported<T>()>* = nullptr>
  std::unique_ptr<column> inclusive_scan(const column_view& input_view,
                                         null_policy,
                                         rmm::cuda_stream_view stream,
                                         rmm::mr::device_memory_resource* mr)
  {
    auto d_input = column_device_view::create(input_view, stream);

    // build indices of the scan operation results
    rmm::device_uvector<size_type> result(input_view.size(), stream);
    thrust::inclusive_scan(rmm::exec_policy(stream),
                           thrust::counting_iterator<size_type>(0),
                           thrust::counting_iterator<size_type>(input_view.size()),
                           result.begin(),
                           string_scan_operator<Op>{*d_input, input_view.has_nulls()});

    // call gather using the indices to build the output column
    return cudf::strings::detail::gather(
      strings_column_view(input_view), result.begin(), result.end(), false, stream, mr);
  }

 public:
  /**
   * @brief Creates a new column from the input column by applying the scan operation
   *
   * @param input Input column view
   * @param null_handling How null row entries are to be processed
   * @param stream CUDA stream used for device memory operations and kernel launches.
   * @param mr Device memory resource used to allocate the returned column's device memory
   * @return
   *
   * @tparam T type of input column
   */
  template <typename T, typename std::enable_if_t<is_supported<T>()>* = nullptr>
  std::unique_ptr<column> operator()(const column_view& input,
                                     null_policy null_handling,
                                     rmm::cuda_stream_view stream,
                                     rmm::mr::device_memory_resource* mr)
  {
    return inclusive_scan<T>(input, null_handling, stream, mr);
  }

  template <typename T, typename... Args>
  std::enable_if_t<!is_supported<T>(), std::unique_ptr<column>> operator()(Args&&...)
  {
    CUDF_FAIL("Non-arithmetic types not supported for inclusive scan");
  }
};

}  // namespace

std::unique_ptr<column> scan_inclusive(
  const column_view& input,
  std::unique_ptr<aggregation> const& agg,
  null_policy null_handling,
  rmm::cuda_stream_view stream,
  rmm::mr::device_memory_resource* mr = rmm::mr::get_current_device_resource())
{
  auto output = scan_agg_dispatch<scan_dispatcher>(input, agg, null_handling, stream, mr);

  if (null_handling == null_policy::EXCLUDE) {
    output->set_null_mask(detail::copy_bitmask(input, stream, mr), input.null_count());
  } else if (input.nullable()) {
    output->set_null_mask(mask_scan(input, scan_type::INCLUSIVE, stream, mr),
                          cudf::UNKNOWN_NULL_COUNT);
  }

  return output;
}
}  // namespace detail
}  // namespace cudf
