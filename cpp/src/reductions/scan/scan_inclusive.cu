#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2021, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "scan.cuh"

#include <cudf/column/column_device_view.cuh>
#include <cudf/column/column_factories.hpp>
#include <cudf/detail/copy.hpp>
#include <cudf/detail/iterator.cuh>
#include <cudf/detail/null_mask.hpp>
#include <cudf/null_mask.hpp>
#include <cudf/reduction.hpp>

#include <rmm/cuda_stream_view.hpp>
#include <rmm/device_uvector.hpp>
#include <rmm/exec_policy.hpp>

#include <thrust/scan.h>

namespace cudf {
namespace detail {

// logical-and scan of the null mask of the input view
rmm::device_buffer mask_scan(const column_view& input_view,
                             cudf::scan_type inclusive,
                             rmm::cuda_stream_view stream,
                             rmm::mr::device_memory_resource* mr)
{
  rmm::device_buffer mask =
    detail::create_null_mask(input_view.size(), mask_state::UNINITIALIZED, stream, mr);
  auto d_input   = column_device_view::create(input_view, stream);
  auto valid_itr = detail::make_validity_iterator(*d_input);

  auto first_null_position = [&] {
    size_type const first_null = thrust::find_if_not(rmm::exec_policy(stream),
                                                     valid_itr,
                                                     valid_itr + input_view.size(),
                                                     thrust::identity<bool>{}) -
                                 valid_itr;
    size_type const exclusive_offset = (inclusive == scan_type::EXCLUSIVE) ? 1 : 0;
    return std::min(input_view.size(), first_null + exclusive_offset);
  }();

  cudf::detail::set_null_mask(
    static_cast<cudf::bitmask_type*>(mask.data()), 0, first_null_position, true, stream);
  cudf::detail::set_null_mask(static_cast<cudf::bitmask_type*>(mask.data()),
                              first_null_position,
                              input_view.size(),
                              false,
                              stream);
  return mask;
}

namespace {

/**
 * @brief Dispatcher for running Scan operation on input column
 *
 * @tparam Op device binary operator
 */
template <typename Op>
struct scan_dispatcher {
 private:
  template <typename T>
  static constexpr bool is_string_supported()
  {
    return std::is_same<T, string_view>::value &&
           (std::is_same<Op, cudf::DeviceMin>::value || std::is_same<Op, cudf::DeviceMax>::value);
  }

  template <typename T>
  static constexpr bool is_supported()
  {
    return std::is_arithmetic<T>::value || is_string_supported<T>();
  }

  // for arithmetic types
  template <typename T, std::enable_if_t<std::is_arithmetic<T>::value>* = nullptr>
  auto inclusive_scan(const column_view& input_view,
                      null_policy,
                      rmm::cuda_stream_view stream,
                      rmm::mr::device_memory_resource* mr)
  {
    auto output_column = detail::allocate_like(
      input_view, input_view.size(), mask_allocation_policy::NEVER, stream, mr);
    mutable_column_view result = output_column->mutable_view();

    auto d_input = column_device_view::create(input_view, stream);
    auto const begin =
      make_null_replacement_iterator(*d_input, Op::template identity<T>(), input_view.has_nulls());
    thrust::inclusive_scan(
      rmm::exec_policy(stream), begin, begin + input_view.size(), result.data<T>(), Op{});

    CHECK_CUDA(stream.value());
    return output_column;
  }

  // for string type: only MIN and MAX are supported
  template <typename T, std::enable_if_t<is_string_supported<T>()>* = nullptr>
  std::unique_ptr<column> inclusive_scan(const column_view& input_view,
                                         null_policy,
                                         rmm::cuda_stream_view stream,
                                         rmm::mr::device_memory_resource* mr)
  {
    auto d_input = column_device_view::create(input_view, stream);

    rmm::device_uvector<T> result(input_view.size(), stream);
    auto begin =
      make_null_replacement_iterator(*d_input, Op::template identity<T>(), input_view.has_nulls());
    thrust::inclusive_scan(
      rmm::exec_policy(stream), begin, begin + input_view.size(), result.data(), Op{});

    CHECK_CUDA(stream.value());
    return cudf::make_strings_column(result, Op::template identity<string_view>(), stream, mr);
  }

 public:
  /**
   * @brief creates new column from input column by applying scan operation
   *
   * @param input     input column view
   * @param inclusive inclusive or exclusive scan
   * @param stream CUDA stream used for device memory operations and kernel launches.
   * @param mr Device memory resource used to allocate the returned column's device memory
   * @return
   *
   * @tparam T type of input column
   */
  template <typename T, typename std::enable_if_t<is_supported<T>()>* = nullptr>
  std::unique_ptr<column> operator()(const column_view& input,
                                     null_policy null_handling,
                                     rmm::cuda_stream_view stream,
                                     rmm::mr::device_memory_resource* mr)
  {
    return inclusive_scan<T>(input, null_handling, stream, mr);
  }

  template <typename T, typename... Args>
  std::enable_if_t<!is_supported<T>(), std::unique_ptr<column>> operator()(Args&&...)
  {
    CUDF_FAIL("Non-arithmetic types not supported for inclusive scan");
  }
};

}  // namespace

std::unique_ptr<column> scan_inclusive(
  const column_view& input,
  std::unique_ptr<aggregation> const& agg,
  null_policy null_handling,
  rmm::cuda_stream_view stream,
  rmm::mr::device_memory_resource* mr = rmm::mr::get_current_device_resource())
{
  auto output = scan_agg_dispatch<scan_dispatcher>(input, agg, null_handling, stream, mr);

  if (null_handling == null_policy::EXCLUDE) {
    output->set_null_mask(detail::copy_bitmask(input, stream, mr), input.null_count());
  } else if (input.nullable()) {
    output->set_null_mask(mask_scan(input, scan_type::INCLUSIVE, stream, mr),
                          cudf::UNKNOWN_NULL_COUNT);
  }

  return output;
}
}  // namespace detail
}  // namespace cudf
