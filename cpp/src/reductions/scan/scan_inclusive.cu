#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2021-2022, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <reductions/scan/scan.cuh>
#include <reductions/struct_minmax_util.cuh>

#include <cudf/column/column_device_view.cuh>
#include <cudf/column/column_factories.hpp>
#include <cudf/detail/copy.hpp>
#include <cudf/detail/gather.hpp>
#include <cudf/detail/iterator.cuh>
#include <cudf/detail/null_mask.hpp>
#include <cudf/reduction.hpp>

#include <rmm/cuda_stream_view.hpp>
#include <rmm/device_uvector.hpp>
#include <rmm/exec_policy.hpp>

#include <thrust/find.h>
#include <thrust/functional.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/scan.h>

#include <type_traits>

namespace cudf {
namespace detail {

// logical-and scan of the null mask of the input view
rmm::device_buffer mask_scan(column_view const& input_view,
                             scan_type inclusive,
                             rmm::cuda_stream_view stream,
                             rmm::mr::device_memory_resource* mr)
{
  rmm::device_buffer mask =
    detail::create_null_mask(input_view.size(), mask_state::UNINITIALIZED, stream, mr);
  auto d_input   = column_device_view::create(input_view, stream);
  auto valid_itr = detail::make_validity_iterator(*d_input);

  auto first_null_position = [&] {
    size_type const first_null =
      thrust::find_if_not(
        rmm::exec_policy(stream), valid_itr, valid_itr + input_view.size(), thrust::identity{}) -
      valid_itr;
    size_type const exclusive_offset = (inclusive == scan_type::EXCLUSIVE) ? 1 : 0;
    return std::min(input_view.size(), first_null + exclusive_offset);
  }();

  set_null_mask(static_cast<bitmask_type*>(mask.data()), 0, first_null_position, true, stream);
  set_null_mask(
    static_cast<bitmask_type*>(mask.data()), first_null_position, input_view.size(), false, stream);
  return mask;
}

namespace {

/**
 * @brief Min/Max inclusive scan operator
 *
 * This operator will accept index values, check them and then
 * run the `Op` operation on the individual element objects.
 * The returned result is the appropriate index value.
 *
 * This was specifically created to workaround a thrust issue
 * https://github.com/NVIDIA/thrust/issues/1479
 * where invalid values are passed to the operator.
 */
template <typename Element, typename Op>
struct min_max_scan_operator {
  column_device_view const col;      ///< strings column device view
  Element const null_replacement{};  ///< value used when element is null
  bool const has_nulls;              ///< true if col has null elements

  min_max_scan_operator(column_device_view const& col, bool has_nulls = true)
    : col{col}, null_replacement{Op::template identity<Element>()}, has_nulls{has_nulls}
  {
    // verify validity bitmask is non-null, otherwise, is_null_nocheck() will crash
    if (has_nulls) CUDF_EXPECTS(col.nullable(), "column with nulls must have a validity bitmask");
  }

  __device__ inline size_type operator()(size_type lhs, size_type rhs) const
  {
    // thrust::inclusive_scan may pass us garbage values so we need to protect ourselves;
    // in these cases the return value does not matter since the result is not used
    if (lhs < 0 || rhs < 0 || lhs >= col.size() || rhs >= col.size()) return 0;
    Element d_lhs =
      has_nulls && col.is_null_nocheck(lhs) ? null_replacement : col.element<Element>(lhs);
    Element d_rhs =
      has_nulls && col.is_null_nocheck(rhs) ? null_replacement : col.element<Element>(rhs);
    return Op{}(d_lhs, d_rhs) == d_lhs ? lhs : rhs;
  }
};

template <typename Op, typename T>
struct scan_functor {
  static std::unique_ptr<column> invoke(column_view const& input_view,
                                        rmm::cuda_stream_view stream,
                                        rmm::mr::device_memory_resource* mr)
  {
    auto output_column = detail::allocate_like(
      input_view, input_view.size(), mask_allocation_policy::NEVER, stream, mr);
    mutable_column_view result = output_column->mutable_view();

    auto d_input = column_device_view::create(input_view, stream);
    auto const begin =
      make_null_replacement_iterator(*d_input, Op::template identity<T>(), input_view.has_nulls());
    thrust::inclusive_scan(
      rmm::exec_policy(stream), begin, begin + input_view.size(), result.data<T>(), Op{});

    CUDF_CHECK_CUDA(stream.value());
    return output_column;
  }
};

template <typename Op>
struct scan_functor<Op, cudf::string_view> {
  static std::unique_ptr<column> invoke(column_view const& input_view,
                                        rmm::cuda_stream_view stream,
                                        rmm::mr::device_memory_resource* mr)
  {
    auto d_input = column_device_view::create(input_view, stream);

    // build indices of the scan operation results
    rmm::device_uvector<size_type> result(input_view.size(), stream);
    thrust::inclusive_scan(
      rmm::exec_policy(stream),
      thrust::counting_iterator<size_type>(0),
      thrust::counting_iterator<size_type>(input_view.size()),
      result.begin(),
      min_max_scan_operator<cudf::string_view, Op>{*d_input, input_view.has_nulls()});

    // call gather using the indices to build the output column
    auto result_table = cudf::detail::gather(cudf::table_view({input_view}),
                                             result,
                                             out_of_bounds_policy::DONT_CHECK,
                                             negative_index_policy::NOT_ALLOWED,
                                             stream,
                                             mr);
    return std::move(result_table->release().front());
  }
};

template <typename Op>
struct scan_functor<Op, cudf::struct_view> {
  static std::unique_ptr<column> invoke(column_view const& input,
                                        rmm::cuda_stream_view stream,
                                        rmm::mr::device_memory_resource* mr)
  {
    // Create a gather map containing indices of the prefix min/max elements.
    auto gather_map = rmm::device_uvector<size_type>(input.size(), stream);
    auto const binop_generator =
      cudf::reduction::detail::comparison_binop_generator::create<Op>(input, stream);
    thrust::inclusive_scan(rmm::exec_policy(stream),
                           thrust::counting_iterator<size_type>(0),
                           thrust::counting_iterator<size_type>(input.size()),
                           gather_map.begin(),
                           binop_generator.binop());

    // Gather the children columns of the input column. Must use `get_sliced_child` to properly
    // handle input in case it is a sliced view.
    auto const input_children = [&] {
      auto const it = cudf::detail::make_counting_transform_iterator(
        0, [structs_view = structs_column_view{input}, stream](auto const child_idx) {
          return structs_view.get_sliced_child(child_idx);
        });
      return std::vector<column_view>(it, it + input.num_children());
    }();

    // Gather the children elements of the prefix min/max struct elements for the output.
    auto scanned_children = cudf::detail::gather(table_view{input_children},
                                                 gather_map,
                                                 out_of_bounds_policy::DONT_CHECK,
                                                 negative_index_policy::NOT_ALLOWED,
                                                 stream,
                                                 mr)
                              ->release();

    // Don't need to set a null mask because that will be handled at the caller.
    return make_structs_column(input.size(),
                               std::move(scanned_children),
                               UNKNOWN_NULL_COUNT,
                               rmm::device_buffer{0, stream, mr},
                               stream,
                               mr);
  }
};

/**
 * @brief Dispatcher for running a Scan operation on an input column
 *
 * @tparam Op device binary operator
 */
template <typename Op>
struct scan_dispatcher {
 private:
  template <typename T>
  static constexpr bool is_supported()
  {
    if constexpr (std::is_same_v<T, cudf::struct_view>) {
      return std::is_same_v<Op, DeviceMin> || std::is_same_v<Op, DeviceMax>;
    } else {
      return std::is_invocable_v<Op, T, T> && !cudf::is_dictionary<T>();
    }
  }

 public:
  /**
   * @brief Creates a new column from the input column by applying the scan operation
   *
   * @param input Input column view
   * @param null_handling How null row entries are to be processed
   * @param stream CUDA stream used for device memory operations and kernel launches.
   * @param mr Device memory resource used to allocate the returned column's device memory
   * @return
   *
   * @tparam T type of input column
   */
  template <typename T, std::enable_if_t<is_supported<T>()>* = nullptr>
  std::unique_ptr<column> operator()(column_view const& input,
                                     null_policy,
                                     rmm::cuda_stream_view stream,
                                     rmm::mr::device_memory_resource* mr)
  {
    return scan_functor<Op, T>::invoke(input, stream, mr);
  }

  template <typename T, typename... Args>
  std::enable_if_t<!is_supported<T>(), std::unique_ptr<column>> operator()(Args&&...)
  {
    CUDF_FAIL("Unsupported type for inclusive scan operation");
  }
};

}  // namespace

std::unique_ptr<column> scan_inclusive(
  column_view const& input,
  scan_aggregation const& agg,
  null_policy null_handling,
  rmm::cuda_stream_view stream,
  rmm::mr::device_memory_resource* mr = rmm::mr::get_current_device_resource())
{
  auto output = scan_agg_dispatch<scan_dispatcher>(input, agg, null_handling, stream, mr);

  if (null_handling == null_policy::EXCLUDE) {
    output->set_null_mask(detail::copy_bitmask(input, stream, mr), input.null_count());
  } else if (input.nullable()) {
    output->set_null_mask(mask_scan(input, scan_type::INCLUSIVE, stream, mr), UNKNOWN_NULL_COUNT);
  }

  // If the input is a structs column, we also need to push down nulls from the parent output column
  // into the children columns.
  if (input.type().id() == type_id::STRUCT && output->has_nulls()) {
    auto const num_rows   = output->size();
    auto const null_count = output->null_count();
    auto content          = output->release();

    // Build new children columns.
    const auto null_mask = reinterpret_cast<bitmask_type const*>(content.null_mask->data());
    std::for_each(content.children.begin(),
                  content.children.end(),
                  [null_mask, null_count, stream, mr](auto& child) {
                    child = structs::detail::superimpose_nulls(
                      null_mask, null_count, std::move(child), stream, mr);
                  });

    // Replace the children columns.
    output = cudf::make_structs_column(
      num_rows, std::move(content.children), null_count, std::move(*content.null_mask), stream, mr);
  }

  return output;
}
}  // namespace detail
}  // namespace cudf
