#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019-2020, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <cudf/detail/reduction_functions.hpp>
#include <cudf/dictionary/dictionary_column_view.hpp>
#include <reductions/simple.cuh>

namespace cudf {
namespace reduction {

std::unique_ptr<cudf::scalar> min(column_view const& col,
                                  data_type const output_dtype,
                                  rmm::cuda_stream_view stream,
                                  rmm::mr::device_memory_resource* mr)
{
  auto const input_type =
    cudf::is_dictionary(col.type()) ? cudf::dictionary_column_view(col).keys().type() : col.type();
  CUDF_EXPECTS(input_type == output_dtype, "min() operation requires matching output type");
  auto const dispatch_type = cudf::is_dictionary(col.type())
                               ? cudf::dictionary_column_view(col).indices().type()
                               : col.type();
  return cudf::type_dispatcher(dispatch_type,
                               simple::same_element_type_dispatcher<cudf::reduction::op::min>{},
                               col,
                               stream,
                               mr);
}

}  // namespace reduction
}  // namespace cudf
