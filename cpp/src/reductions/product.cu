/*
 * Copyright (c) 2019, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */
 // The translation unit for reduction `product`

#include "reduction_functions.cuh"
#include "simple.cuh"

gdf_scalar cudf::reduction::product(gdf_column const& col, gdf_dtype const output_dtype, hipStream_t stream)
{
    using reducer = cudf::reduction::simple::element_type_dispatcher<cudf::reduction::op::product>;
    return cudf::type_dispatcher(col.dtype, reducer(), col, output_dtype, stream);
}


