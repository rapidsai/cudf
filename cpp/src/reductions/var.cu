/*
 * Copyright (c) 2019, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

// The translation unit for reduction `variance`

#include <cudf/detail/reduction_functions.hpp>
#include "compound.cuh"

// @param[in] ddof Delta Degrees of Freedom used for `std`, `var`.
//                 The divisor used in calculations is N - ddof, where N
//                 represents the number of elements.

std::unique_ptr<cudf::scalar> cudf::experimental::reduction::variance(
  column_view const& col,
  cudf::data_type const output_dtype,
  cudf::size_type ddof,
  rmm::mr::device_memory_resource* mr,
  hipStream_t stream)
{
  // TODO: add cuda version check when the fix is available
#if !defined(__CUDACC_DEBUG__)
  using reducer = cudf::experimental::reduction::compound::element_type_dispatcher<
    cudf::experimental::reduction::op::variance>;
  return cudf::experimental::type_dispatcher(
    col.type(), reducer(), col, output_dtype, ddof, mr, stream);
#else
  // workaround for bug 200529165 which causes compilation error only at device
  // debug build the bug will be fixed at cuda 10.2
  CUDF_FAIL("var/std reductions are not supported at debug build.");
#endif
}
