#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2023-2024, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <cudf/column/column_factories.hpp>
#include <cudf/detail/gather.hpp>
#include <cudf/detail/hash_reduce_by_row.cuh>
#include <cudf/detail/iterator.cuh>
#include <cudf/scalar/scalar.hpp>
#include <cudf/structs/structs_column_view.hpp>

#include <rmm/resource_ref.hpp>

#include <cuda/atomic>
#include <cuda/functional>
#include <thrust/copy.h>
#include <thrust/iterator/zip_iterator.h>
#include <thrust/tuple.h>

#include <optional>

namespace cudf::reduction::detail {

namespace {

// Always use 64-bit signed integer for storing count.
using histogram_count_type = int64_t;

/**
 * @brief The functor to accumulate the frequency of each distinct rows in the input table.
 */
template <typename MapView, typename KeyHasher, typename KeyEqual, typename CountType>
struct reduce_fn : cudf::detail::reduce_by_row_fn_base<MapView, KeyHasher, KeyEqual, CountType> {
  CountType const* d_partial_output;

  reduce_fn(MapView const& d_map,
            KeyHasher const& d_hasher,
            KeyEqual const& d_equal,
            CountType* const d_output,
            CountType const* const d_partial_output)
    : cudf::detail::reduce_by_row_fn_base<MapView, KeyHasher, KeyEqual, CountType>{d_map,
                                                                                   d_hasher,
                                                                                   d_equal,
                                                                                   d_output},
      d_partial_output{d_partial_output}
  {
  }

  // Count the number of rows in each group of rows that are compared equal.
  __device__ void operator()(size_type const idx) const
  {
    auto const increment = d_partial_output ? d_partial_output[idx] : CountType{1};
    auto const count =
      cuda::atomic_ref<CountType, cuda::thread_scope_device>(*this->get_output_ptr(idx));
    count.fetch_add(increment, cuda::std::memory_order_relaxed);
  }
};

/**
 * @brief The builder to construct an instance of `reduce_fn` functor.
 */
template <typename CountType>
struct reduce_func_builder {
  CountType const* const d_partial_output;

  reduce_func_builder(CountType const* const d_partial_output) : d_partial_output{d_partial_output}
  {
  }

  template <typename MapView, typename KeyHasher, typename KeyEqual>
  auto build(MapView const& d_map,
             KeyHasher const& d_hasher,
             KeyEqual const& d_equal,
             CountType* const d_output)
  {
    return reduce_fn<MapView, KeyHasher, KeyEqual, CountType>{
      d_map, d_hasher, d_equal, d_output, d_partial_output};
  }
};

/**
 * @brief Specialized functor to check for not-zero of the second component of the input.
 */
struct is_not_zero {
  template <typename Pair>
  __device__ bool operator()(Pair const input) const
  {
    return thrust::get<1>(input) != 0;
  }
};

/**
 * @brief Building a histogram by gathering distinct rows from the input table and their
 * corresponding distinct counts.
 *
 * @param input The input table
 * @param distinct_indices Indices of the distinct rows
 * @param distinct_counts Distinct counts corresponding to the distinct rows
 * @param stream CUDA stream used for device memory operations and kernel launches
 * @param mr Device memory resource used to allocate the returned object's device memory
 * @return A list_scalar storing the output histogram
 */
auto gather_histogram(table_view const& input,
                      device_span<size_type const> distinct_indices,
                      std::unique_ptr<column>&& distinct_counts,
                      rmm::cuda_stream_view stream,
                      rmm::device_async_resource_ref mr)
{
  auto distinct_rows = cudf::detail::gather(input,
                                            distinct_indices,
                                            out_of_bounds_policy::DONT_CHECK,
                                            cudf::detail::negative_index_policy::NOT_ALLOWED,
                                            stream,
                                            mr);

  std::vector<std::unique_ptr<column>> struct_children;
  struct_children.emplace_back(std::move(distinct_rows->release().front()));
  struct_children.emplace_back(std::move(distinct_counts));
  auto output_structs = make_structs_column(
    static_cast<size_type>(distinct_indices.size()), std::move(struct_children), 0, {}, stream, mr);

  return std::make_unique<cudf::list_scalar>(
    std::move(*output_structs.release()), true, stream, mr);
}

}  // namespace

std::unique_ptr<column> make_empty_histogram_like(column_view const& values)
{
  std::vector<std::unique_ptr<column>> struct_children;
  struct_children.emplace_back(empty_like(values));
  struct_children.emplace_back(make_numeric_column(data_type{type_id::INT64}, 0));
  return std::make_unique<column>(data_type{type_id::STRUCT},
                                  0,
                                  rmm::device_buffer{},
                                  rmm::device_buffer{},
                                  0,
                                  std::move(struct_children));
}

std::pair<std::unique_ptr<rmm::device_uvector<size_type>>, std::unique_ptr<column>>
compute_row_frequencies(table_view const& input,
                        std::optional<column_view> const& partial_counts,
                        rmm::cuda_stream_view stream,
                        rmm::device_async_resource_ref mr)
{
  auto const has_nested_columns = cudf::detail::has_nested_columns(input);

  // Nested types are not tested, thus we just throw exception if we see such input for now.
  // We should remove this check after having enough tests.
  CUDF_EXPECTS(!has_nested_columns,
               "Nested types are not yet supported in histogram aggregation.",
               std::invalid_argument);

  auto map = cudf::detail::hash_map_type{compute_hash_table_size(input.num_rows()),
                                         cuco::empty_key{-1},
                                         cuco::empty_value{std::numeric_limits<size_type>::min()},
                                         cudf::detail::cuco_allocator{stream},
                                         stream.value()};

  auto const preprocessed_input =
    cudf::experimental::row::hash::preprocessed_table::create(input, stream);
  auto const has_nulls = nullate::DYNAMIC{cudf::has_nested_nulls(input)};

  auto const row_hasher = cudf::experimental::row::hash::row_hasher(preprocessed_input);
  auto const key_hasher = row_hasher.device_hasher(has_nulls);
  auto const row_comp   = cudf::experimental::row::equality::self_comparator(preprocessed_input);

  auto const pair_iter = cudf::detail::make_counting_transform_iterator(
    size_type{0},
    cuda::proclaim_return_type<cuco::pair<size_type, size_type>>(
      [] __device__(size_type const i) { return cuco::make_pair(i, i); }));

  // Always compare NaNs as equal.
  using nan_equal_comparator =
    cudf::experimental::row::equality::nan_equal_physical_equality_comparator;
  auto const value_comp = nan_equal_comparator{};

  if (has_nested_columns) {
    auto const key_equal = row_comp.equal_to<true>(has_nulls, null_equality::EQUAL, value_comp);
    map.insert(pair_iter, pair_iter + input.num_rows(), key_hasher, key_equal, stream.value());
  } else {
    auto const key_equal = row_comp.equal_to<false>(has_nulls, null_equality::EQUAL, value_comp);
    map.insert(pair_iter, pair_iter + input.num_rows(), key_hasher, key_equal, stream.value());
  }

  // Gather the indices of distinct rows.
  auto distinct_indices = std::make_unique<rmm::device_uvector<size_type>>(
    static_cast<size_type>(map.get_size()), stream, mr);

  // Store the number of occurrences of each distinct row.
  auto distinct_counts = make_numeric_column(data_type{type_to_id<histogram_count_type>()},
                                             static_cast<size_type>(map.get_size()),
                                             mask_state::UNALLOCATED,
                                             stream,
                                             mr);

  // Compute frequencies (aka distinct counts) for the input rows.
  // Note that we consider null and NaNs as always equal.
  auto const reduction_results = cudf::detail::hash_reduce_by_row(
    map,
    preprocessed_input,
    input.num_rows(),
    has_nulls,
    has_nested_columns,
    null_equality::EQUAL,
    nan_equality::ALL_EQUAL,
    reduce_func_builder<histogram_count_type>{
      partial_counts ? partial_counts.value().begin<histogram_count_type>() : nullptr},
    histogram_count_type{0},
    stream,
    rmm::mr::get_current_device_resource());

  auto const input_it = thrust::make_zip_iterator(
    thrust::make_tuple(thrust::make_counting_iterator(0), reduction_results.begin()));
  auto const output_it = thrust::make_zip_iterator(thrust::make_tuple(
    distinct_indices->begin(), distinct_counts->mutable_view().begin<histogram_count_type>()));

  // Reduction results above are either group sizes of equal rows, or `0`.
  // The final output is non-zero group sizes only.
  thrust::copy_if(
    rmm::exec_policy(stream), input_it, input_it + input.num_rows(), output_it, is_not_zero{});

  return {std::move(distinct_indices), std::move(distinct_counts)};
}

std::unique_ptr<cudf::scalar> histogram(column_view const& input,
                                        rmm::cuda_stream_view stream,
                                        rmm::device_async_resource_ref mr)
{
  // Empty group should be handled before reaching here.
  CUDF_EXPECTS(input.size() > 0, "Input should not be empty.", std::invalid_argument);

  auto const input_tv = table_view{{input}};
  auto [distinct_indices, distinct_counts] =
    compute_row_frequencies(input_tv, std::nullopt, stream, mr);
  return gather_histogram(input_tv, *distinct_indices, std::move(distinct_counts), stream, mr);
}

std::unique_ptr<cudf::scalar> merge_histogram(column_view const& input,
                                              rmm::cuda_stream_view stream,
                                              rmm::device_async_resource_ref mr)
{
  // Empty group should be handled before reaching here.
  CUDF_EXPECTS(input.size() > 0, "Input should not be empty.", std::invalid_argument);
  CUDF_EXPECTS(!input.has_nulls(), "The input column must not have nulls.", std::invalid_argument);
  CUDF_EXPECTS(input.type().id() == type_id::STRUCT && input.num_children() == 2,
               "The input must be a structs column having two children.",
               std::invalid_argument);
  CUDF_EXPECTS(cudf::is_integral(input.child(1).type()) && !input.child(1).has_nulls(),
               "The second child of the input column must be of integral type and without nulls.",
               std::invalid_argument);

  auto const structs_cv   = structs_column_view{input};
  auto const input_values = structs_cv.get_sliced_child(0, stream);
  auto const input_counts = structs_cv.get_sliced_child(1, stream);

  auto const values_tv = table_view{{input_values}};
  auto [distinct_indices, distinct_counts] =
    compute_row_frequencies(values_tv, input_counts, stream, mr);
  return gather_histogram(values_tv, *distinct_indices, std::move(distinct_counts), stream, mr);
}

}  // namespace cudf::reduction::detail
