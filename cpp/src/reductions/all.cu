#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019-2021, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <cudf/detail/reduction_functions.hpp>
#include <cudf/detail/utilities/device_atomics.cuh>
#include <cudf/dictionary/dictionary_column_view.hpp>
#include <reductions/simple.cuh>

namespace cudf {
namespace reduction {
namespace detail {
namespace {

/**
 * @brief Compute reduction all() for dictionary columns.
 *
 * This compiles 10x faster than using thrust::reduce or the
 * cudf::simple::reduction::detail::reduce utility.
 * Both of these use the CUB DeviceReduce which aggressively inlines
 * the input iterator logic.
 */
struct all_fn {
  template <typename Iterator>
  struct all_true_fn {
    __device__ void operator()(size_type idx)
    {
      if (*d_result && (iter[idx] != *d_result)) atomicAnd(d_result, false);
    }
    Iterator iter;
    bool* d_result;
  };

  template <typename T, std::enable_if_t<std::is_arithmetic<T>::value>* = nullptr>
  std::unique_ptr<scalar> operator()(column_view const& input,
                                     rmm::cuda_stream_view stream,
                                     rmm::mr::device_memory_resource* mr)
  {
    auto const d_dict = cudf::column_device_view::create(input, stream);
    auto const iter   = [&] {
      auto null_iter =
        cudf::reduction::op::min{}.template get_null_replacing_element_transformer<bool>();
      auto pair_iter =
        cudf::dictionary::detail::make_dictionary_pair_iterator<T>(*d_dict, input.has_nulls());
      return thrust::make_transform_iterator(pair_iter, null_iter);
    }();
    auto result = std::make_unique<numeric_scalar<bool>>(true, true, stream, mr);
    thrust::for_each_n(rmm::exec_policy(stream),
                       thrust::make_counting_iterator<size_type>(0),
                       input.size(),
                       all_true_fn<decltype(iter)>{iter, result->data()});
    return result;
  }
  template <typename T, std::enable_if_t<!std::is_arithmetic<T>::value>* = nullptr>
  std::unique_ptr<scalar> operator()(column_view const&,
                                     rmm::cuda_stream_view,
                                     rmm::mr::device_memory_resource*)
  {
    CUDF_FAIL("Unexpected key type for dictionary in reduction all()");
  }
};

}  // namespace
}  // namespace detail

std::unique_ptr<cudf::scalar> all(column_view const& col,
                                  cudf::data_type const output_dtype,
                                  rmm::cuda_stream_view stream,
                                  rmm::mr::device_memory_resource* mr)
{
  CUDF_EXPECTS(output_dtype == cudf::data_type(cudf::type_id::BOOL8),
               "all() operation can be applied with output type `BOOL8` only");

  if (cudf::is_dictionary(col.type())) {
    return cudf::type_dispatcher(
      dictionary_column_view(col).keys().type(), detail::all_fn{}, col, stream, mr);
  }
  // dispatch for non-dictionary types
  return cudf::type_dispatcher(col.type(),
                               simple::bool_result_element_dispatcher<cudf::reduction::op::min>{},
                               col,
                               stream,
                               mr);
}

}  // namespace reduction
}  // namespace cudf
