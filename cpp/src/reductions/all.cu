#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019-2020, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <reductions/simple.cuh>

#include <cudf/detail/reduction_functions.hpp>

#include <rmm/cuda_stream_view.hpp>

namespace cudf {
namespace reduction {

std::unique_ptr<cudf::scalar> all(column_view const& col,
                                  cudf::data_type const output_dtype,
                                  rmm::cuda_stream_view stream,
                                  rmm::mr::device_memory_resource* mr)
{
  CUDF_EXPECTS(output_dtype == cudf::data_type(cudf::type_id::BOOL8),
               "all() operation can be applied with output type `BOOL8` only");
  return cudf::type_dispatcher(col.type(),
                               simple::bool_result_element_dispatcher<cudf::reduction::op::min>{},
                               col,
                               stream,
                               mr);
}

}  // namespace reduction
}  // namespace cudf
