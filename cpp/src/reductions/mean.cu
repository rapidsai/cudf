/*
 * Copyright (c) 2019-2024, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "compound.cuh"

#include <cudf/dictionary/dictionary_column_view.hpp>
#include <cudf/reduction/detail/reduction_functions.hpp>
#include <cudf/utilities/memory_resource.hpp>

#include <rmm/cuda_stream_view.hpp>

namespace cudf {
namespace reduction {
namespace detail {

std::unique_ptr<cudf::scalar> mean(column_view const& col,
                                   cudf::data_type const output_dtype,
                                   rmm::cuda_stream_view stream,
                                   rmm::device_async_resource_ref mr)
{
  auto col_type =
    cudf::is_dictionary(col.type()) ? dictionary_column_view(col).keys().type() : col.type();

  using reducer = compound::detail::element_type_dispatcher<op::mean>;
  return cudf::type_dispatcher(
    col_type, reducer(), col, output_dtype, /* ddof is not used for mean*/ 1, stream, mr);
}

}  // namespace detail
}  // namespace reduction
}  // namespace cudf
