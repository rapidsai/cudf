/*
 * Copyright (c) 2019, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */
// The translation unit for reduction `mean`

#include <cudf/detail/reduction_functions.hpp>
#include "compound.cuh"

std::unique_ptr<cudf::scalar> cudf::experimental::reduction::mean(
  column_view const& col,
  cudf::data_type const output_dtype,
  rmm::mr::device_memory_resource* mr,
  hipStream_t stream)
{
  using reducer = cudf::experimental::reduction::compound::element_type_dispatcher<
    cudf::experimental::reduction::op::mean>;
  return cudf::experimental::type_dispatcher(
    col.type(), reducer(), col, output_dtype, /* ddof is not used for mean*/ 1, mr, stream);
}
