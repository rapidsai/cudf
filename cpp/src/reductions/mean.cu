/*
 * Copyright (c) 2019, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */
 // The translation unit for reduction `mean`

#include "reduction_functions.cuh"
#include "compound.cuh"

gdf_scalar cudf::reduction::mean(gdf_column const& col, gdf_dtype const output_dtype, hipStream_t stream)
{
    using reducer = cudf::reduction::compound::element_type_dispatcher<cudf::reduction::op::mean>;
    return cudf::type_dispatcher(col.dtype, reducer(), col, output_dtype,  /* ddof is not used for mean*/ 1, stream);
}


