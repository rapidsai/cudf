/*
 * Copyright (c) 2020, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <cudf/column/column_factories.hpp>
#include <cudf/detail/concatenate.cuh>
#include <cudf/detail/stream_compaction.hpp>
#include <cudf/dictionary/detail/concatenate.hpp>
#include <cudf/dictionary/dictionary_column_view.hpp>
#include <cudf/dictionary/dictionary_factories.hpp>
#include <cudf/table/table.hpp>
#include <cudf/table/table_view.hpp>

#include <thrust/binary_search.h>
#include <thrust/transform_scan.h>
#include <algorithm>
#include <rmm/device_uvector.hpp>
#include <vector>

namespace cudf {
namespace dictionary {
namespace detail {
namespace {

/**
 * @brief Keys and indices offsets values.
 *
 * The first value is the keys offsets and the second values is the indices offsets.
 * These are offsets to the beginning of each input column after concatenating.
 */
using offsets_pair = thrust::pair<size_type, size_type>;

/**
 * @brief Utility for calculating the offsets for the concatenated child columns
 *        of the output dictionary column.
 */
struct compute_children_offsets_fn {
  /**
   * @brief Create the utility functor.
   *
   * The columns vector is converted into vector of column_view pointers so they
   * can be used in thrust::transform_exclusive_scan without causing the
   * compiler warning/error: "host/device function calling host function".
   *
   * @param columns The input dictionary columns.
   */
  compute_children_offsets_fn(std::vector<column_view> const& columns)
    : columns_ptrs{columns.size()}
  {
    std::transform(
      columns.begin(), columns.end(), columns_ptrs.begin(), [](auto& cv) { return &cv; });
  }

  /**
   * @brief Return the first keys().type of the dictionary columns.
   */
  data_type get_keys_type()
  {
    auto const view(*std::find_if(
      columns_ptrs.begin(), columns_ptrs.end(), [](auto pcv) { return pcv->size() > 0; }));
    return dictionary_column_view(*view).keys().type();
  }

  /**
   * @brief Create the offsets pair for the concatenated columns.
   *
   * Both vectors have the length of the number of input columns.
   * The sizes of each child (keys and indices) of the individual columns
   * are used to create the offsets.
   *
   * @param stream Stream used for allocating the output rmm::device_uvector.
   * @return Vector of offsets_pair objects for keys and indices.
   */
  rmm::device_uvector<offsets_pair> create_children_offsets(hipStream_t stream)
  {
    std::vector<offsets_pair> offsets(columns_ptrs.size());
    thrust::transform_exclusive_scan(
      thrust::host,
      columns_ptrs.begin(),
      columns_ptrs.end(),
      offsets.begin(),
      [](auto pcv) {
        dictionary_column_view view(*pcv);
        return offsets_pair{view.keys_size(), view.size()};
      },
      offsets_pair{0, 0},
      [](auto lhs, auto rhs) {
        return offsets_pair{lhs.first + rhs.first, lhs.second + rhs.second};
      });
    auto d_offsets = rmm::device_uvector<offsets_pair>(offsets.size(), stream);
    CUDA_TRY(hipMemcpyAsync(d_offsets.data(),
                             offsets.data(),
                             offsets.size() * sizeof(offsets_pair),
                             hipMemcpyHostToDevice,
                             stream));
    CUDA_TRY(hipStreamSynchronize(stream));
    return d_offsets;
  }

 private:
  std::vector<column_view const*> columns_ptrs;  ///< pointer version of input column_view vector
};

/**
 * @brief Type-dispatch functor for remapping the old indices to new values based
 * on the new key-set.
 *
 * The dispatch is based on the key type.
 * The output column is the updated indices child for the new dictionary column.
 */
struct dispatch_compute_indices {
  template <typename Element>
  typename std::enable_if_t<cudf::is_relationally_comparable<Element, Element>(),
                            std::unique_ptr<column>>
  operator()(column_view const& all_keys,
             column_view const& all_indices,
             column_view const& new_keys,
             offsets_pair const* d_offsets,
             size_type const* d_map_to_keys,
             hipStream_t stream,
             rmm::mr::device_memory_resource* mr)
  {
    auto keys_view     = column_device_view::create(all_keys, stream);
    auto indices_view  = column_device_view::create(all_indices, stream);
    auto d_all_indices = *indices_view;

    // map the concatenated indices to the concatenated keys
    auto all_itr = thrust::make_permutation_iterator(
      keys_view->begin<Element>(),
      thrust::make_transform_iterator(
        thrust::make_counting_iterator<size_type>(0),
        [d_offsets, d_map_to_keys, d_all_indices] __device__(size_type idx) {
          if (d_all_indices.is_null(idx)) return 0;
          return d_all_indices.template element<int32_t>(idx) + d_offsets[d_map_to_keys[idx]].first;
        }));

    auto new_keys_view = column_device_view::create(new_keys, stream);
    // create the indices output column
    auto result = make_numeric_column(
      all_indices.type(), all_indices.size(), mask_state::UNALLOCATED, stream, mr);
    auto d_result = result->mutable_view().data<int32_t>();
    // new indices values are computed by matching the concatenated keys to the new key set
    thrust::lower_bound(rmm::exec_policy(stream)->on(stream),
                        new_keys_view->begin<Element>(),
                        new_keys_view->end<Element>(),
                        all_itr,
                        all_itr + all_indices.size(),
                        d_result,
                        thrust::less<Element>());
    return result;
  }

  template <typename Element>
  typename std::enable_if_t<!cudf::is_relationally_comparable<Element, Element>(),
                            std::unique_ptr<column>>
  operator()(column_view const&,
             column_view const&,
             column_view const&,
             offsets_pair const*,
             size_type const*,
             hipStream_t stream,
             rmm::mr::device_memory_resource*)
  {
    CUDF_FAIL("list_view as keys for dictionary not supported");
  }
};

}  // namespace

std::unique_ptr<column> concatenate(std::vector<column_view> const& columns,
                                    hipStream_t stream,
                                    rmm::mr::device_memory_resource* mr)
{
  // exception here is the same behavior as in cudf::concatenate
  CUDF_EXPECTS(not columns.empty(), "Unexpected empty list of columns to concatenate.");

  // concatenate the keys (and check the keys match)
  compute_children_offsets_fn child_offsets_fn{columns};
  auto keys_type = child_offsets_fn.get_keys_type();
  std::vector<column_view> keys_views(columns.size());
  std::transform(columns.begin(), columns.end(), keys_views.begin(), [keys_type](auto cv) {
    auto dict_view = dictionary_column_view(cv);
    // empty column may not have keys so we create an empty column_view place-holder
    if (dict_view.size() == 0) return column_view{keys_type, 0, nullptr};
    auto keys = dict_view.keys();
    CUDF_EXPECTS(keys.type() == keys_type, "key types of all dictionary columns must match");
    return keys;
  });
  auto all_keys =
    cudf::detail::concatenate(keys_views, rmm::mr::get_current_device_resource(), stream);

  // sort keys and remove duplicates;
  // this becomes the keys child for the output dictionary column
  auto table_keys = cudf::detail::drop_duplicates(table_view{{all_keys->view()}},
                                                  std::vector<size_type>{0},
                                                  duplicate_keep_option::KEEP_FIRST,
                                                  null_equality::EQUAL,
                                                  mr,
                                                  stream)
                      ->release();
  std::unique_ptr<column> keys_column(std::move(table_keys.front()));

  // next, concatenate the indices
  std::vector<column_view> indices_views(columns.size());
  std::transform(columns.begin(), columns.end(), indices_views.begin(), [](auto cv) {
    auto dict_view = dictionary_column_view(cv);
    if (dict_view.size() == 0) return column_view{data_type{type_id::UINT32}, 0, nullptr};
    return dict_view.get_indices_annotated();  // nicely includes validity mask and view offset
  });
  auto all_indices        = cudf::detail::concatenate(indices_views, mr, stream);
  auto const indices_size = all_indices->size();

  // build a vector of values to map the old indices to the concatenated keys
  auto children_offsets = child_offsets_fn.create_children_offsets(stream);
  rmm::device_uvector<size_type> map_to_keys(indices_size, stream);
  auto indices_itr = thrust::make_transform_iterator(thrust::make_counting_iterator<size_type>(1),
                                                     [] __device__(size_type idx) {
                                                       return offsets_pair{0, idx};
                                                     });
  // the indices offsets (pair.second) are for building the map
  thrust::lower_bound(
    rmm::exec_policy(stream)->on(stream),
    children_offsets.begin() + 1,
    children_offsets.end(),
    indices_itr,
    indices_itr + indices_size + 1,
    map_to_keys.begin(),
    [] __device__(auto const& lhs, auto const& rhs) { return lhs.second < rhs.second; });

  // now recompute the indices values for the new keys_column;
  // the keys offsets (pair.first) are for mapping to the input keys
  auto indices_column = type_dispatcher(keys_type,
                                        dispatch_compute_indices{},
                                        all_keys->view(),     // old keys
                                        all_indices->view(),  // old indices
                                        keys_column->view(),  // new keys
                                        children_offsets.data(),
                                        map_to_keys.data(),
                                        stream,
                                        mr);

  // remove the bitmask from the all_indices
  auto null_count = all_indices->null_count();  // get before release()
  auto contents   = all_indices->release();     // all_indices will now be empty

  // finally, frankenstein that dictionary column together
  return make_dictionary_column(std::move(keys_column),
                                std::move(indices_column),
                                std::move(*(contents.null_mask.release())),
                                null_count);
}

}  // namespace detail
}  // namespace dictionary
}  // namespace cudf
