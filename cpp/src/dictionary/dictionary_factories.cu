/*
 * Copyright (c) 2020, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <cudf/column/column_factories.hpp>
#include <cudf/column/column_view.hpp>
#include <cudf/dictionary/dictionary_factories.hpp>

namespace cudf {
std::unique_ptr<column> make_dictionary_column(column_view const& keys_column,
                                               column_view const& indices_column,
                                               rmm::mr::device_memory_resource* mr,
                                               hipStream_t stream)
{
  CUDF_EXPECTS(!keys_column.has_nulls(), "keys column must not have nulls");
  if (keys_column.size() == 0) return make_empty_column(data_type{type_id::DICTIONARY32});
  CUDF_EXPECTS(indices_column.type().id() == cudf::type_id::INT32, "indices column must be INT32");

  auto keys_copy = std::make_unique<column>(keys_column, stream, mr);
  column_view indices_view{indices_column.type(),
                           indices_column.size(),
                           indices_column.data<int32_t>(),
                           nullptr,
                           0,
                           indices_column.offset()};
  auto indices_copy = std::make_unique<column>(indices_view, stream, mr);
  rmm::device_buffer null_mask{0, stream, mr};
  auto null_count = indices_column.null_count();
  if (null_count) null_mask = copy_bitmask(indices_column, stream, mr);

  std::vector<std::unique_ptr<column>> children;
  children.emplace_back(std::move(indices_copy));
  children.emplace_back(std::move(keys_copy));
  return std::make_unique<column>(data_type{type_id::DICTIONARY32},
                                  indices_column.size(),
                                  rmm::device_buffer{0, stream, mr},
                                  std::move(null_mask),
                                  null_count,
                                  std::move(children));
}

std::unique_ptr<column> make_dictionary_column(std::unique_ptr<column> keys_column,
                                               std::unique_ptr<column> indices_column,
                                               rmm::device_buffer&& null_mask,
                                               size_type null_count)
{
  CUDF_EXPECTS(!keys_column->has_nulls(), "keys column must not have nulls");
  CUDF_EXPECTS(!indices_column->has_nulls(), "indices column must not have nulls");
  CUDF_EXPECTS(indices_column->type().id() == cudf::type_id::INT32, "indices must be type INT32");

  auto count = indices_column->size();
  std::vector<std::unique_ptr<column>> children;
  children.emplace_back(std::move(indices_column));
  children.emplace_back(std::move(keys_column));
  return std::make_unique<column>(data_type{type_id::DICTIONARY32},
                                  count,
                                  rmm::device_buffer{},
                                  std::move(null_mask),
                                  null_count,
                                  std::move(children));
}

}  // namespace cudf
