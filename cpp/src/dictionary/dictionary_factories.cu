/*
 * Copyright (c) 2020, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <cudf/column/column_factories.hpp>
#include <cudf/column/column_view.hpp>
#include <cudf/dictionary/dictionary_factories.hpp>
#include <cudf/utilities/type_dispatcher.hpp>

namespace cudf {
namespace {
struct dispatch_create_indices {
  template <typename IndexType, std::enable_if_t<is_index_type<IndexType>()>* = nullptr>
  std::unique_ptr<column> operator()(column_view const& indices,
                                     rmm::mr::device_memory_resource* mr,
                                     hipStream_t stream)
  {
    CUDF_EXPECTS(std::is_unsigned<IndexType>(), "indices must be an unsigned type");
    column_view indices_view{
      indices.type(), indices.size(), indices.data<IndexType>(), nullptr, 0, indices.offset()};
    return std::make_unique<column>(indices_view, stream, mr);
  }
  template <typename IndexType, std::enable_if_t<!is_index_type<IndexType>()>* = nullptr>
  std::unique_ptr<column> operator()(column_view const&,
                                     rmm::mr::device_memory_resource*,
                                     hipStream_t)
  {
    CUDF_FAIL("indices must be an integer type.");
  }
};
}  // namespace

std::unique_ptr<column> make_dictionary_column(column_view const& keys_column,
                                               column_view const& indices_column,
                                               rmm::mr::device_memory_resource* mr,
                                               hipStream_t stream)
{
  CUDF_EXPECTS(!keys_column.has_nulls(), "keys column must not have nulls");
  if (keys_column.size() == 0) return make_empty_column(data_type{type_id::DICTIONARY32});

  auto keys_copy = std::make_unique<column>(keys_column, stream, mr);
  auto indices_copy =
    type_dispatcher(indices_column.type(), dispatch_create_indices{}, indices_column, mr, stream);
  rmm::device_buffer null_mask{0, stream, mr};
  auto null_count = indices_column.null_count();
  if (null_count) null_mask = copy_bitmask(indices_column, stream, mr);

  std::vector<std::unique_ptr<column>> children;
  children.emplace_back(std::move(indices_copy));
  children.emplace_back(std::move(keys_copy));
  return std::make_unique<column>(data_type{type_id::DICTIONARY32},
                                  indices_column.size(),
                                  rmm::device_buffer{0, stream, mr},
                                  std::move(null_mask),
                                  null_count,
                                  std::move(children));
}

std::unique_ptr<column> make_dictionary_column(std::unique_ptr<column> keys_column,
                                               std::unique_ptr<column> indices_column,
                                               rmm::device_buffer&& null_mask,
                                               size_type null_count)
{
  CUDF_EXPECTS(!keys_column->has_nulls(), "keys column must not have nulls");
  CUDF_EXPECTS(!indices_column->has_nulls(), "indices column must not have nulls");
  CUDF_EXPECTS(is_unsigned(indices_column->type()), "indices must be type unsigned integer");

  auto count = indices_column->size();
  std::vector<std::unique_ptr<column>> children;
  children.emplace_back(std::move(indices_column));
  children.emplace_back(std::move(keys_column));
  return std::make_unique<column>(data_type{type_id::DICTIONARY32},
                                  count,
                                  rmm::device_buffer{},
                                  std::move(null_mask),
                                  null_count,
                                  std::move(children));
}

}  // namespace cudf
