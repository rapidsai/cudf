/*
 * Copyright (c) 2020, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <cudf/column/column_factories.hpp>
#include <cudf/column/column_view.hpp>
#include <cudf/detail/unary.hpp>
#include <cudf/dictionary/detail/encode.hpp>
#include <cudf/dictionary/dictionary_factories.hpp>
#include <cudf/utilities/type_dispatcher.hpp>

namespace cudf {
namespace {
struct dispatch_create_indices {
  template <typename IndexType, std::enable_if_t<is_index_type<IndexType>()>* = nullptr>
  std::unique_ptr<column> operator()(column_view const& indices,
                                     rmm::mr::device_memory_resource* mr,
                                     hipStream_t stream)
  {
    CUDF_EXPECTS(std::is_unsigned<IndexType>(), "indices must be an unsigned type");
    column_view indices_view{
      indices.type(), indices.size(), indices.data<IndexType>(), nullptr, 0, indices.offset()};
    return std::make_unique<column>(indices_view, stream, mr);
  }
  template <typename IndexType, std::enable_if_t<!is_index_type<IndexType>()>* = nullptr>
  std::unique_ptr<column> operator()(column_view const&,
                                     rmm::mr::device_memory_resource*,
                                     hipStream_t)
  {
    CUDF_FAIL("indices must be an integer type.");
  }
};
}  // namespace

std::unique_ptr<column> make_dictionary_column(column_view const& keys_column,
                                               column_view const& indices_column,
                                               rmm::mr::device_memory_resource* mr,
                                               hipStream_t stream)
{
  CUDF_EXPECTS(!keys_column.has_nulls(), "keys column must not have nulls");
  if (keys_column.size() == 0) return make_empty_column(data_type{type_id::DICTIONARY32});

  auto keys_copy = std::make_unique<column>(keys_column, stream, mr);
  auto indices_copy =
    type_dispatcher(indices_column.type(), dispatch_create_indices{}, indices_column, mr, stream);
  rmm::device_buffer null_mask{0, stream, mr};
  auto null_count = indices_column.null_count();
  if (null_count) null_mask = copy_bitmask(indices_column, stream, mr);

  std::vector<std::unique_ptr<column>> children;
  children.emplace_back(std::move(indices_copy));
  children.emplace_back(std::move(keys_copy));
  return std::make_unique<column>(data_type{type_id::DICTIONARY32},
                                  indices_column.size(),
                                  rmm::device_buffer{0, stream, mr},
                                  std::move(null_mask),
                                  null_count,
                                  std::move(children));
}

std::unique_ptr<column> make_dictionary_column(std::unique_ptr<column> keys_column,
                                               std::unique_ptr<column> indices_column,
                                               rmm::device_buffer&& null_mask,
                                               size_type null_count)
{
  CUDF_EXPECTS(!keys_column->has_nulls(), "keys column must not have nulls");
  CUDF_EXPECTS(!indices_column->has_nulls(), "indices column must not have nulls");
  CUDF_EXPECTS(is_unsigned(indices_column->type()), "indices must be type unsigned integer");

  auto count = indices_column->size();
  std::vector<std::unique_ptr<column>> children;
  children.emplace_back(std::move(indices_column));
  children.emplace_back(std::move(keys_column));
  return std::make_unique<column>(data_type{type_id::DICTIONARY32},
                                  count,
                                  rmm::device_buffer{},
                                  std::move(null_mask),
                                  null_count,
                                  std::move(children));
}

namespace {

/**
 * @brief This functor maps signed type_ids to unsigned counterparts.
 */
struct make_unsigned_fn {
  template <typename T, std::enable_if_t<is_index_type<T>()>* = nullptr>
  constexpr cudf::type_id operator()()
  {
    return cudf::type_to_id<std::make_unsigned_t<T>>();
  }
  template <typename T, std::enable_if_t<not is_index_type<T>()>* = nullptr>
  constexpr cudf::type_id operator()()
  {
    return cudf::type_to_id<T>();
  }
};

}  // namespace

std::unique_ptr<column> make_dictionary_column(std::unique_ptr<column> keys,
                                               std::unique_ptr<column> indices,
                                               rmm::mr::device_memory_resource* mr,
                                               hipStream_t stream)
{
  CUDF_EXPECTS(!keys->has_nulls(), "keys column must not have nulls");

  // signed integer data can be used directly in the unsigned indices column
  auto const indices_type = cudf::type_dispatcher(indices->type(), make_unsigned_fn{});
  auto const indices_size = indices->size();        // these need to be saved
  auto const null_count   = indices->null_count();  // before calling release()
  auto contents           = indices->release();
  // compute the indices type using the size of the key set
  auto const new_type = dictionary::detail::get_indices_type_for_size(keys->size());

  // create the dictionary indices: convert to unsigned and remove nulls
  auto indices_column = [&] {
    // If the types match, then just commandeer the column's data buffer.
    if (new_type.id() == indices_type) {
      return std::make_unique<column>(
        new_type, indices_size, *(contents.data.release()), rmm::device_buffer{0, stream, mr}, 0);
    }
    // If the new type does not match, then convert the data.
    cudf::column_view cast_view{cudf::data_type{indices_type}, indices_size, contents.data->data()};
    return cudf::detail::cast(cast_view, new_type, mr, stream);
  }();

  return make_dictionary_column(std::move(keys),
                                std::move(indices_column),
                                std::move(*(contents.null_mask.release())),
                                null_count);
}

}  // namespace cudf
