/*
 * Copyright (c) 2020, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <cudf/column/column_device_view.cuh>
#include <cudf/column/column_factories.hpp>
#include <cudf/detail/concatenate.hpp>
#include <cudf/detail/indexalator.cuh>
#include <cudf/detail/nvtx/ranges.hpp>
#include <cudf/detail/search.hpp>
#include <cudf/detail/stream_compaction.hpp>
#include <cudf/detail/valid_if.cuh>
#include <cudf/dictionary/detail/encode.hpp>
#include <cudf/dictionary/dictionary_column_view.hpp>
#include <cudf/dictionary/dictionary_factories.hpp>
#include <cudf/stream_compaction.hpp>

#include <rmm/thrust_rmm_allocator.h>
#include <thrust/binary_search.h>

namespace cudf {
namespace dictionary {
namespace detail {
namespace {
/**
 * @brief Type-dispatch functor for remapping the old indices to new values based on the new
 * key-set.
 *
 * The dispatch is based on the key type.
 * The output column is the new indices column for the new dictionary column.
 */
struct dispatch_compute_indices {
  template <typename Element>
  typename std::enable_if_t<cudf::is_relationally_comparable<Element, Element>(),
                            std::unique_ptr<column>>
  operator()(dictionary_column_view const& input,
             column_view const& new_keys,
             rmm::mr::device_memory_resource* mr,
             hipStream_t stream)
  {
    auto dictionary_view = column_device_view::create(input.parent(), stream);
    auto d_dictionary    = *dictionary_view;
    auto keys_view       = column_device_view::create(input.keys(), stream);
    auto dictionary_itr  = thrust::make_permutation_iterator(
      keys_view->begin<Element>(),
      thrust::make_transform_iterator(
        thrust::make_counting_iterator<size_type>(0), [d_dictionary] __device__(size_type idx) {
          if (d_dictionary.is_null(idx)) return 0;
          return static_cast<size_type>(d_dictionary.element<dictionary32>(idx));
        }));
    auto new_keys_view = column_device_view::create(new_keys, stream);

    // create output indices column
    auto result = make_numeric_column(get_indices_type_for_size(new_keys.size()),
                                      input.size(),
                                      mask_state::UNALLOCATED,
                                      stream,
                                      mr);
    auto result_itr =
      cudf::detail::indexalator_factory::make_output_iterator(result->mutable_view());
    thrust::lower_bound(rmm::exec_policy(stream)->on(stream),
                        new_keys_view->begin<Element>(),
                        new_keys_view->end<Element>(),
                        dictionary_itr,
                        dictionary_itr + input.size(),
                        result_itr,
                        thrust::less<Element>());
    result->set_null_count(0);
    return result;
  }

  template <typename Element>
  typename std::enable_if_t<!cudf::is_relationally_comparable<Element, Element>(),
                            std::unique_ptr<column>>
  operator()(dictionary_column_view const& input,
             column_view const& new_keys,
             rmm::mr::device_memory_resource* mr,
             hipStream_t stream)
  {
    CUDF_FAIL("list_view dictionary set_keys not supported yet");
  }
};

}  // namespace

//
std::unique_ptr<column> set_keys(
  dictionary_column_view const& dictionary_column,
  column_view const& new_keys,
  rmm::mr::device_memory_resource* mr = rmm::mr::get_current_device_resource(),
  hipStream_t stream                 = 0)
{
  CUDF_EXPECTS(!new_keys.has_nulls(), "keys parameter must not have nulls");
  auto keys = dictionary_column.keys();
  CUDF_EXPECTS(keys.type() == new_keys.type(), "keys types must match");

  // copy the keys -- use drop_duplicates to make sure they are sorted and unique
  auto table_keys = cudf::detail::drop_duplicates(table_view{{new_keys}},
                                                  std::vector<size_type>{0},
                                                  duplicate_keep_option::KEEP_FIRST,
                                                  null_equality::EQUAL,
                                                  mr,
                                                  stream)
                      ->release();
  std::unique_ptr<column> keys_column(std::move(table_keys.front()));

  // compute the new nulls
  auto matches   = cudf::detail::contains(keys, keys_column->view(), mr, stream);
  auto d_matches = matches->view().data<bool>();
  auto indices_itr =
    cudf::detail::indexalator_factory::make_input_iterator(dictionary_column.indices());
  auto d_null_mask = dictionary_column.null_mask();
  auto new_nulls   = cudf::detail::valid_if(
    thrust::make_counting_iterator<size_type>(dictionary_column.offset()),
    thrust::make_counting_iterator<size_type>(dictionary_column.offset() +
                                              dictionary_column.size()),
    [d_null_mask, indices_itr, d_matches] __device__(size_type idx) {
      if (d_null_mask && !bit_is_set(d_null_mask, idx)) return false;
      return d_matches[indices_itr[idx]];
    },
    stream,
    mr);

  // compute the new indices
  auto indices_column = type_dispatcher(keys_column->type(),
                                        dispatch_compute_indices{},
                                        dictionary_column,
                                        keys_column->view(),
                                        mr,
                                        stream);

  // create column with keys_column and indices_column
  return make_dictionary_column(std::move(keys_column),
                                std::move(indices_column),
                                std::move(new_nulls.first),
                                new_nulls.second);
}

std::vector<std::unique_ptr<column>> match_dictionaries(std::vector<dictionary_column_view> input,
                                                        rmm::mr::device_memory_resource* mr,
                                                        hipStream_t stream)
{
  std::vector<column_view> keys(input.size());
  std::transform(input.begin(), input.end(), keys.begin(), [](auto& col) { return col.keys(); });
  auto new_keys  = cudf::detail::concatenate(keys, rmm::mr::get_current_device_resource(), stream);
  auto keys_view = new_keys->view();
  std::vector<std::unique_ptr<column>> result(input.size());
  std::transform(input.begin(), input.end(), result.begin(), [keys_view, mr, stream](auto& col) {
    return set_keys(col, keys_view, mr, stream);
  });
  return result;
}

}  // namespace detail

// external API

std::unique_ptr<column> set_keys(dictionary_column_view const& dictionary_column,
                                 column_view const& keys,
                                 rmm::mr::device_memory_resource* mr)
{
  CUDF_FUNC_RANGE();
  return detail::set_keys(dictionary_column, keys, mr);
}

}  // namespace dictionary
}  // namespace cudf
