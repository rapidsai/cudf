/*
 * Copyright (c) 2020, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <cudf/column/column_device_view.cuh>
#include <cudf/column/column_factories.hpp>
#include <cudf/detail/nvtx/ranges.hpp>
#include <cudf/detail/search.hpp>
#include <cudf/detail/stream_compaction.hpp>
#include <cudf/detail/valid_if.cuh>
#include <cudf/dictionary/dictionary_column_view.hpp>
#include <cudf/dictionary/dictionary_factories.hpp>
#include <cudf/stream_compaction.hpp>

#include <rmm/thrust_rmm_allocator.h>
#include <thrust/binary_search.h>

namespace cudf {
namespace dictionary {
namespace detail {
namespace {
/**
 * @brief Type-dispatch functor for remapping the old indices to new values based on the new
 * key-set.
 *
 * The dispatch is based on the key type.
 * The output column is the new indices column for the new dictionary column.
 */
struct dispatch_compute_indices {
  template <typename Element>
  typename std::enable_if_t<cudf::is_relationally_comparable<Element, Element>(),
                            std::unique_ptr<column>>
  operator()(dictionary_column_view const& input,
             column_view const& new_keys,
             rmm::mr::device_memory_resource* mr,
             hipStream_t stream)
  {
    auto dictionary_view = column_device_view::create(input.parent(), stream);
    auto d_dictionary    = *dictionary_view;
    auto dictionary_itr  = thrust::make_transform_iterator(
      thrust::make_counting_iterator<size_type>(0), [d_dictionary] __device__(size_type idx) {
        if (d_dictionary.is_null(idx)) return Element{};
        column_device_view d_keys = d_dictionary.child(1);
        size_type index           = static_cast<size_type>(d_dictionary.element<dictionary32>(idx));
        return d_keys.template element<Element>(index);
      });
    auto new_keys_view = column_device_view::create(new_keys, stream);
    auto d_new_keys    = *new_keys_view;
    auto keys_itr      = thrust::make_transform_iterator(
      thrust::make_counting_iterator<size_type>(0),
      [d_new_keys] __device__(size_type idx) { return d_new_keys.template element<Element>(idx); });

    auto result = make_numeric_column(
      data_type{type_id::INT32}, input.size(), mask_state::UNALLOCATED, stream, mr);
    auto d_result = result->mutable_view().data<int32_t>();
    auto execpol  = rmm::exec_policy(stream);
    thrust::lower_bound(execpol->on(stream),
                        keys_itr,
                        keys_itr + new_keys.size(),
                        dictionary_itr,
                        dictionary_itr + input.size(),
                        d_result,
                        thrust::less<Element>());
    result->set_null_count(0);
    return result;
  }

  template <typename Element>
  typename std::enable_if_t<!cudf::is_relationally_comparable<Element, Element>(),
                            std::unique_ptr<column>>
  operator()(dictionary_column_view const& input,
             column_view const& new_keys,
             rmm::mr::device_memory_resource* mr,
             hipStream_t stream)
  {
    CUDF_FAIL("list_view dictionary set_keys not supported yet");
  }
};

}  // namespace

//
std::unique_ptr<column> set_keys(
  dictionary_column_view const& dictionary_column,
  column_view const& new_keys,
  rmm::mr::device_memory_resource* mr = rmm::mr::get_default_resource(),
  hipStream_t stream                 = 0)
{
  CUDF_EXPECTS(!new_keys.has_nulls(), "keys parameter must not have nulls");
  auto keys = dictionary_column.keys();
  CUDF_EXPECTS(keys.type() == new_keys.type(), "keys types must match");

  // copy the keys -- use drop_duplicates to make sure they are sorted and unique
  auto table_keys = cudf::detail::drop_duplicates(table_view{{new_keys}},
                                                  std::vector<size_type>{0},
                                                  duplicate_keep_option::KEEP_FIRST,
                                                  null_equality::EQUAL,
                                                  mr,
                                                  stream)
                      ->release();
  std::unique_ptr<column> keys_column(std::move(table_keys.front()));

  // compute the new nulls
  auto matches     = cudf::detail::contains(keys, keys_column->view(), mr, stream);
  auto d_matches   = matches->view().data<bool>();
  auto d_indices   = dictionary_column.indices().data<int32_t>();
  auto d_null_mask = dictionary_column.null_mask();
  auto new_nulls   = cudf::detail::valid_if(
    thrust::make_counting_iterator<size_type>(dictionary_column.offset()),
    thrust::make_counting_iterator<size_type>(dictionary_column.offset() +
                                              dictionary_column.size()),
    [d_null_mask, d_indices, d_matches] __device__(size_type idx) {
      if (d_null_mask && !bit_is_set(d_null_mask, idx)) return false;
      return d_matches[d_indices[idx]];
    },
    stream,
    mr);

  // compute the new indices
  auto indices_column = type_dispatcher(keys_column->type(),
                                        dispatch_compute_indices{},
                                        dictionary_column,
                                        keys_column->view(),
                                        mr,
                                        stream);

  // create column with keys_column and indices_column
  return make_dictionary_column(std::move(keys_column),
                                std::move(indices_column),
                                std::move(new_nulls.first),
                                new_nulls.second);
}
}  // namespace detail

// external API

std::unique_ptr<column> set_keys(dictionary_column_view const& dictionary_column,
                                 column_view const& keys,
                                 rmm::mr::device_memory_resource* mr)
{
  CUDF_FUNC_RANGE();
  return detail::set_keys(dictionary_column, keys, mr);
}

}  // namespace dictionary
}  // namespace cudf
