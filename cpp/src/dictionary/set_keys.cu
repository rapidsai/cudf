/*
 * Copyright (c) 2020, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <cudf/column/column_device_view.cuh>
#include <cudf/column/column_factories.hpp>
#include <cudf/detail/concatenate.hpp>
#include <cudf/detail/indexalator.cuh>
#include <cudf/detail/nvtx/ranges.hpp>
#include <cudf/detail/search.hpp>
#include <cudf/detail/stream_compaction.hpp>
#include <cudf/detail/valid_if.cuh>
#include <cudf/dictionary/detail/encode.hpp>
#include <cudf/dictionary/dictionary_column_view.hpp>
#include <cudf/dictionary/dictionary_factories.hpp>
#include <cudf/stream_compaction.hpp>

#include <rmm/thrust_rmm_allocator.h>
#include <thrust/binary_search.h>
#include <algorithm>
#include <iterator>

namespace cudf {
namespace dictionary {
namespace detail {
namespace {
/**
 * @brief Type-dispatch functor for remapping the old indices to new values based on the new
 * key-set.
 *
 * The dispatch is based on the key type.
 * The output column is the new indices column for the new dictionary column.
 */
struct dispatch_compute_indices {
  template <typename Element>
  typename std::enable_if_t<cudf::is_relationally_comparable<Element, Element>(),
                            std::unique_ptr<column>>
  operator()(dictionary_column_view const& input,
             column_view const& new_keys,
             rmm::mr::device_memory_resource* mr,
             hipStream_t stream)
  {
    auto dictionary_view = column_device_view::create(input.parent(), stream);
    auto d_dictionary    = *dictionary_view;
    auto keys_view       = column_device_view::create(input.keys(), stream);
    auto dictionary_itr  = thrust::make_permutation_iterator(
      keys_view->begin<Element>(),
      thrust::make_transform_iterator(
        thrust::make_counting_iterator<size_type>(0), [d_dictionary] __device__(size_type idx) {
          if (d_dictionary.is_null(idx)) return 0;
          return static_cast<size_type>(d_dictionary.element<dictionary32>(idx));
        }));
    auto new_keys_view = column_device_view::create(new_keys, stream);

    // create output indices column
    auto result = make_numeric_column(get_indices_type_for_size(new_keys.size()),
                                      input.size(),
                                      mask_state::UNALLOCATED,
                                      stream,
                                      mr);
    auto result_itr =
      cudf::detail::indexalator_factory::make_output_iterator(result->mutable_view());
    thrust::lower_bound(rmm::exec_policy(stream)->on(stream),
                        new_keys_view->begin<Element>(),
                        new_keys_view->end<Element>(),
                        dictionary_itr,
                        dictionary_itr + input.size(),
                        result_itr,
                        thrust::less<Element>());
    result->set_null_count(0);
    return result;
  }

  template <typename Element>
  typename std::enable_if_t<!cudf::is_relationally_comparable<Element, Element>(),
                            std::unique_ptr<column>>
  operator()(dictionary_column_view const& input,
             column_view const& new_keys,
             rmm::mr::device_memory_resource* mr,
             hipStream_t stream)
  {
    CUDF_FAIL("list_view dictionary set_keys not supported yet");
  }
};

}  // namespace

//
std::unique_ptr<column> set_keys(
  dictionary_column_view const& dictionary_column,
  column_view const& new_keys,
  rmm::mr::device_memory_resource* mr = rmm::mr::get_current_device_resource(),
  hipStream_t stream                 = 0)
{
  CUDF_EXPECTS(!new_keys.has_nulls(), "keys parameter must not have nulls");
  auto keys = dictionary_column.keys();
  CUDF_EXPECTS(keys.type() == new_keys.type(), "keys types must match");

  // copy the keys -- use drop_duplicates to make sure they are sorted and unique
  auto table_keys = cudf::detail::drop_duplicates(table_view{{new_keys}},
                                                  std::vector<size_type>{0},
                                                  duplicate_keep_option::KEEP_FIRST,
                                                  null_equality::EQUAL,
                                                  stream,
                                                  mr)
                      ->release();
  std::unique_ptr<column> keys_column(std::move(table_keys.front()));

  // compute the new nulls
  auto matches   = cudf::detail::contains(keys, keys_column->view(), stream, mr);
  auto d_matches = matches->view().data<bool>();
  auto indices_itr =
    cudf::detail::indexalator_factory::make_input_iterator(dictionary_column.indices());
  auto d_null_mask = dictionary_column.null_mask();
  auto new_nulls   = cudf::detail::valid_if(
    thrust::make_counting_iterator<size_type>(dictionary_column.offset()),
    thrust::make_counting_iterator<size_type>(dictionary_column.offset() +
                                              dictionary_column.size()),
    [d_null_mask, indices_itr, d_matches] __device__(size_type idx) {
      if (d_null_mask && !bit_is_set(d_null_mask, idx)) return false;
      return d_matches[indices_itr[idx]];
    },
    stream,
    mr);

  // compute the new indices
  auto indices_column = type_dispatcher(keys_column->type(),
                                        dispatch_compute_indices{},
                                        dictionary_column,
                                        keys_column->view(),
                                        mr,
                                        stream);

  // create column with keys_column and indices_column
  return make_dictionary_column(std::move(keys_column),
                                std::move(indices_column),
                                std::move(new_nulls.first),
                                new_nulls.second);
}

std::vector<std::unique_ptr<column>> match_dictionaries(std::vector<dictionary_column_view> input,
                                                        rmm::mr::device_memory_resource* mr,
                                                        hipStream_t stream)
{
  std::vector<column_view> keys(input.size());
  std::transform(input.begin(), input.end(), keys.begin(), [](auto& col) { return col.keys(); });
  auto new_keys  = cudf::detail::concatenate(keys, stream);
  auto keys_view = new_keys->view();
  std::vector<std::unique_ptr<column>> result(input.size());
  std::transform(input.begin(), input.end(), result.begin(), [keys_view, mr, stream](auto& col) {
    return set_keys(col, keys_view, mr, stream);
  });
  return result;
}

std::pair<std::vector<std::unique_ptr<column>>, std::vector<table_view>> match_dictionaries(
  std::vector<table_view> tables, rmm::mr::device_memory_resource* mr, hipStream_t stream)
{
  // Make a copy of all the column views from each table_view
  std::vector<std::vector<column_view>> updated_columns;
  std::transform(tables.begin(), tables.end(), std::back_inserter(updated_columns), [](auto& t) {
    return std::vector<column_view>(t.begin(), t.end());
  });

  // Each column in a table must match in type.
  // Once a dictionary column is found, all the corresponding column_views in the
  // other table_views are matched. The matched column_views then replace the originals.
  std::vector<std::unique_ptr<column>> dictionary_columns;
  auto first_table = tables.front();
  for (size_type col_idx = 0; col_idx < first_table.num_columns(); ++col_idx) {
    auto col = first_table.column(col_idx);
    if (col.type().id() == type_id::DICTIONARY32) {
      std::vector<dictionary_column_view> dict_views;  // hold all column_views at col_idx
      std::transform(
        tables.begin(), tables.end(), std::back_inserter(dict_views), [col_idx](auto& t) {
          return dictionary_column_view(t.column(col_idx));
        });
      // now match the keys in these dictionary columns
      auto dict_cols = dictionary::detail::match_dictionaries(dict_views, mr, stream);
      // replace the updated_columns vector entries for the set of columns at col_idx
      auto dict_col_idx = 0;
      for (auto& v : updated_columns) v[col_idx] = dict_cols[dict_col_idx++]->view();
      // move the updated dictionary columns into the main output vector
      std::move(dict_cols.begin(), dict_cols.end(), std::back_inserter(dictionary_columns));
    }
  }
  // All the new column_views are in now updated_columns.

  // Rebuild the table_views from the column_views.
  std::vector<table_view> updated_tables;
  std::transform(updated_columns.begin(),
                 updated_columns.end(),
                 std::back_inserter(updated_tables),
                 [](auto& v) { return table_view{v}; });

  // Return the new dictionary columns and table_views
  return {std::move(dictionary_columns), std::move(updated_tables)};
}

}  // namespace detail

// external API

std::unique_ptr<column> set_keys(dictionary_column_view const& dictionary_column,
                                 column_view const& keys,
                                 rmm::mr::device_memory_resource* mr)
{
  CUDF_FUNC_RANGE();
  return detail::set_keys(dictionary_column, keys, mr);
}

}  // namespace dictionary
}  // namespace cudf
