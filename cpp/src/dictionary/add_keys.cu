/*
 * Copyright (c) 2020, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <cudf/detail/concatenate.cuh>
#include <cudf/detail/gather.hpp>
#include <cudf/detail/nvtx/ranges.hpp>
#include <cudf/detail/search.hpp>
#include <cudf/detail/stream_compaction.hpp>
#include <cudf/dictionary/dictionary_factories.hpp>
#include <cudf/dictionary/update_keys.hpp>
#include <cudf/stream_compaction.hpp>
#include <cudf/table/table.hpp>
#include <cudf/table/table_view.hpp>

#include <rmm/thrust_rmm_allocator.h>

namespace cudf {
namespace dictionary {
namespace detail {
/**
 * @brief Create a new dictionary column by adding the new keys elements
 * to the existing dictionary_column.
 *
 * ```
 * Example:
 * d1 = {[a, b, c, d, f], {4, 0, 3, 1, 2, 2, 2, 4, 0}}
 * d2 = add_keys( d1, [d, b, e] )
 * d2 is now {[a, b, c, d, e, f], [5, 0, 3, 1, 2, 2, 2, 5, 0]}
 * ```
 *
 */
std::unique_ptr<column> add_keys(
  dictionary_column_view const& dictionary_column,
  column_view const& new_keys,
  rmm::mr::device_memory_resource* mr = rmm::mr::get_current_device_resource(),
  hipStream_t stream                 = 0)
{
  CUDF_EXPECTS(!new_keys.has_nulls(), "Keys must not have nulls");
  auto old_keys = dictionary_column.keys();  // [a,b,c,d,f]
  CUDF_EXPECTS(new_keys.type() == old_keys.type(), "Keys must be the same type");
  // first, concatenate the keys together
  // [a,b,c,d,f] + [d,b,e] = [a,b,c,d,f,d,b,e]
  auto combined_keys = cudf::detail::concatenate(
    std::vector<column_view>{old_keys, new_keys}, rmm::mr::get_current_device_resource(), stream);
  // sort and remove any duplicates from the combined keys
  // drop_duplicates([a,b,c,d,f,d,b,e]) = [a,b,c,d,e,f]
  auto table_keys = cudf::detail::drop_duplicates(table_view{{*combined_keys}},
                                                  std::vector<size_type>{0},  // only one key column
                                                  duplicate_keep_option::KEEP_FIRST,
                                                  null_equality::EQUAL,
                                                  mr,
                                                  stream)
                      ->release();
  std::unique_ptr<column> keys_column(std::move(table_keys.front()));
  // create a map for the indices
  // lower_bound([a,b,c,d,e,f],[a,b,c,d,f]) = [0,1,2,3,5]
  auto map_indices = cudf::detail::lower_bound(
    table_view{{keys_column->view()}},
    table_view{{old_keys}},
    std::vector<order>{order::ASCENDING},
    std::vector<null_order>{null_order::AFTER},  // should be no nulls here
    mr,
    stream);
  // now create the indices column -- map old values to the new ones
  // gather([4,0,3,1,2,2,2,4,0],[0,1,2,3,5]) = [5,0,3,1,2,2,2,5,0]
  column_view indices_view(dictionary_column.indices().type(),
                           dictionary_column.size(),
                           dictionary_column.indices().head(),
                           nullptr,
                           0,
                           dictionary_column.offset());
  // the result may contain nulls if the input contains nulls
  // and the corresponding index is therefore invalid/undefined
  auto table_indices = cudf::detail::gather(table_view{{map_indices->view()}},
                                            indices_view,
                                            cudf::detail::out_of_bounds_policy::IGNORE,
                                            cudf::detail::negative_index_policy::NOT_ALLOWED,
                                            mr,
                                            stream)
                         ->release();
  // The output of lower_bound is INT32 but we need to convert to UINT32.
  // There are no negative values and uint32 is the same width as int32
  // so we can create a UINT32 column using the same data.
  auto const indices_size = static_cast<size_type>(table_indices.front()->size());
  auto contents           = table_indices.front()->release();
  auto indices_column     = std::make_unique<column>(data_type{type_id::UINT32},
                                                 indices_size,
                                                 std::move(*(contents.data.release())),
                                                 rmm::device_buffer{0, stream, mr},
                                                 0);

  // create new dictionary column with keys_column and indices_column
  // null mask has not changed
  return make_dictionary_column(std::move(keys_column),
                                std::move(indices_column),
                                copy_bitmask(dictionary_column.parent(), stream, mr),
                                dictionary_column.null_count());
}

}  // namespace detail

std::unique_ptr<column> add_keys(dictionary_column_view const& dictionary_column,
                                 column_view const& keys,
                                 rmm::mr::device_memory_resource* mr)
{
  CUDF_FUNC_RANGE();
  return detail::add_keys(dictionary_column, keys, mr);
}

}  // namespace dictionary
}  // namespace cudf
