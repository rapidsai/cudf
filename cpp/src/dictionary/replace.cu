/*
 * Copyright (c) 2020, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <cudf/detail/copy.hpp>
#include <cudf/detail/copy_if_else.cuh>
#include <cudf/detail/indexalator.cuh>
#include <cudf/detail/unary.hpp>
#include <cudf/dictionary/detail/encode.hpp>
#include <cudf/dictionary/detail/replace.hpp>
#include <cudf/dictionary/detail/search.hpp>
#include <cudf/dictionary/detail/update_keys.hpp>
#include <cudf/dictionary/dictionary_factories.hpp>

#include <thrust/iterator/constant_iterator.h>
#include <thrust/iterator/counting_iterator.h>

namespace cudf {
namespace dictionary {
namespace detail {
namespace {

/**
 * @brief An index accessor that returns a validity flag along with the index value.
 *
 * This is used to make a `pair_iterator` for calling `copy_if_else`.
 */
template <bool has_nulls = false>
struct nullable_index_accessor {
  cudf::detail::input_indexalator iter;
  bitmask_type const* null_mask{};
  size_type const offset{};

  /**
   * @brief Create an accessor from a column_view.
   */
  nullable_index_accessor(column_view const& col) : null_mask{col.null_mask()}, offset{col.offset()}
  {
    if (has_nulls) { CUDF_EXPECTS(col.nullable(), "Unexpected non-nullable column."); }
    iter = cudf::detail::indexalator_factory::make_input_iterator(col);
  }

  /**
   * @brief Create an accessor from a scalar.
   */
  nullable_index_accessor(scalar const& input)
  {
    iter = cudf::detail::indexalator_factory::make_input_iterator(input);
  }

  __device__ thrust::pair<size_type, bool> operator()(size_type i) const
  {
    return {iter[i], (has_nulls ? bit_is_set(null_mask, i + offset) : true)};
  }
};

/**
 * @brief Create an index iterator with a nullable index accessor.
 */
template <bool has_nulls>
auto make_nullable_index_iterator(column_view const& col)
{
  return thrust::make_transform_iterator(thrust::make_counting_iterator<size_type>(0),
                                         nullable_index_accessor<has_nulls>{col});
}

/**
 * @brief Create an index iterator with a nullable index accessor for a scalar.
 */
auto make_scalar_iterator(scalar const& input)
{
  return thrust::make_transform_iterator(thrust::make_constant_iterator<size_type>(0),
                                         nullable_index_accessor<false>{input});
}

/**
 * @brief This utility uses `copy_if_else` to replace null entries using the input bitmask as a
 * predicate.
 *
 * The predicate identifies which column row to copy from and the bitmask specifies which rows
 * are null. Since the `copy_if_else` accepts iterators, we also supply it with pair-iterators
 * created from indexalators and the validity masks.
 *
 * @tparam ReplacementItr must be a pair iterator of (index,valid).
 *
 * @param input lhs for `copy_if_else`
 * @param replacement_iter rhs for `copy_if_else`
 * @param mr Device memory resource used to allocate the returned column's device memory.
 * @param stream CUDA stream used for device memory operations and kernel launches.
 * @return Always returns column of type INT32 (size_type)
 */
template <typename ReplacementIter>
std::unique_ptr<column> replace_indices(column_view const& input,
                                        ReplacementIter replacement_iter,
                                        rmm::mr::device_memory_resource* mr,
                                        hipStream_t stream)
{
  auto const input_view = column_device_view::create(input, stream);
  auto const d_input    = *input_view;
  auto predicate        = [d_input] __device__(auto i) { return d_input.is_valid(i); };

  auto input_pair_iterator = make_nullable_index_iterator<true>(input);
  return cudf::detail::copy_if_else(true,
                                    input_pair_iterator,
                                    input_pair_iterator + input.size(),
                                    replacement_iter,
                                    predicate,
                                    mr,
                                    stream);
}

}  // namespace

/**
 * @copydoc cudf::dictionary::detail::replace_nulls(cudf::column_view const&,cudf::column_view
 * const&,rmm::mr::device_memory_resource*,hipStream_t)
 */
std::unique_ptr<column> replace_nulls(dictionary_column_view const& input,
                                      dictionary_column_view const& replacement,
                                      rmm::mr::device_memory_resource* mr,
                                      hipStream_t stream)
{
  if (input.is_empty()) { return cudf::empty_like(input.parent()); }
  if (!input.has_nulls()) { return std::make_unique<cudf::column>(input.parent()); }
  CUDF_EXPECTS(input.keys().type() == replacement.keys().type(), "keys must match");
  CUDF_EXPECTS(replacement.size() == input.size(), "column sizes must match");

  // first combine the keys so both input dictionaries have the same set
  auto matched = match_dictionaries({input, replacement}, mr, stream);

  // now build the new indices by doing replace-null using the updated input indices
  auto const input_indices =
    dictionary_column_view(matched.front()->view()).get_indices_annotated();
  auto const repl_indices = dictionary_column_view(matched.back()->view()).get_indices_annotated();
  auto new_indices =
    repl_indices.has_nulls()
      ? replace_indices(input_indices, make_nullable_index_iterator<true>(repl_indices), mr, stream)
      : replace_indices(
          input_indices, make_nullable_index_iterator<false>(repl_indices), mr, stream);

  // auto keys_column = ;
  return make_dictionary_column(
    std::move(matched.front()->release().children.back()), std::move(new_indices), mr, stream);
}

/**
 * @copydoc cudf::dictionary::detail::replace_nulls(cudf::column_view const&,cudf::scalar
 * const&,rmm::mr::device_memory_resource*,hipStream_t)
 */
std::unique_ptr<column> replace_nulls(dictionary_column_view const& input,
                                      scalar const& replacement,
                                      rmm::mr::device_memory_resource* mr,
                                      hipStream_t stream)
{
  if (input.is_empty()) { return cudf::empty_like(input.parent()); }
  if (!input.has_nulls() || !replacement.is_valid()) {
    return std::make_unique<cudf::column>(input.parent());
  }
  CUDF_EXPECTS(input.keys().type() == replacement.type(), "keys must match scalar type");

  // first add the replacment to the keys so only the indices need to be processed
  auto const default_mr = rmm::mr::get_current_device_resource();
  auto input_matched    = dictionary::detail::add_keys(
    input, make_column_from_scalar(replacement, 1, default_mr, stream)->view(), mr, stream);
  auto const input_view   = dictionary_column_view(input_matched->view());
  auto const scalar_index = get_index(input_view, replacement, default_mr, stream);

  // now build the new indices by doing replace-null on the updated indices
  auto const input_indices = input_view.get_indices_annotated();
  auto new_indices =
    replace_indices(input_indices, make_scalar_iterator(*scalar_index), mr, stream);
  new_indices->set_null_mask(rmm::device_buffer{0, stream, mr}, 0);

  return make_dictionary_column(
    std::move(input_matched->release().children.back()), std::move(new_indices), mr, stream);
}

}  // namespace detail
}  // namespace dictionary
}  // namespace cudf
