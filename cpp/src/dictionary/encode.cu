#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2020, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <cudf/column/column.hpp>
#include <cudf/copying.hpp>
#include <cudf/detail/null_mask.hpp>
#include <cudf/detail/nvtx/ranges.hpp>
#include <cudf/detail/transform.hpp>
#include <cudf/detail/unary.hpp>
#include <cudf/dictionary/detail/encode.hpp>
#include <cudf/dictionary/dictionary_factories.hpp>
#include <cudf/dictionary/encode.hpp>
#include <cudf/stream_compaction.hpp>
#include <cudf/table/table.hpp>
#include <cudf/table/table_view.hpp>
#include <rmm/cuda_stream_view.hpp>

namespace cudf {
namespace dictionary {
namespace detail {
/**
 * @copydoc cudf::dictionary::encode
 *
 * @param stream CUDA stream used for device memory operations and kernel launches.
 */
std::unique_ptr<column> encode(column_view const& input_column,
                               data_type indices_type,
                               rmm::mr::device_memory_resource* mr,
                               hipStream_t stream)
{
  CUDF_EXPECTS(is_unsigned(indices_type), "indices must be type unsigned integer");
  CUDF_EXPECTS(input_column.type().id() != type_id::DICTIONARY32,
               "cannot encode a dictionary from a dictionary");

  auto codified       = cudf::detail::encode(cudf::table_view({input_column}), stream, mr);
  auto keys_table     = std::move(codified.first);
  auto indices_column = std::move(codified.second);
  auto keys_column    = std::move(keys_table->release().front());

  if (keys_column->has_nulls()) {
    keys_column = std::make_unique<column>(
      slice(keys_column->view(), std::vector<size_type>{0, keys_column->size() - 1}).front(),
      stream,
      mr);
    keys_column->set_null_mask(rmm::device_buffer{0, stream, mr}, 0);  // remove the null-mask
  }

  // the encode() returns INT32 for indices
  if (indices_column->type().id() != indices_type.id())
    indices_column = cudf::detail::cast(indices_column->view(), indices_type, stream, mr);

  // create column with keys_column and indices_column
  return make_dictionary_column(
    std::move(keys_column),
    std::move(indices_column),
    cudf::detail::copy_bitmask(input_column, rmm::cuda_stream_view{stream}, mr),
    input_column.null_count());
}

/**
 * @copydoc cudf::dictionary::detail::get_indices_type_for_size
 */
data_type get_indices_type_for_size(size_type keys_size)
{
  if (keys_size <= std::numeric_limits<uint8_t>::max()) return data_type{type_id::UINT8};
  if (keys_size <= std::numeric_limits<uint16_t>::max()) return data_type{type_id::UINT16};
  return data_type{type_id::UINT32};
}

}  // namespace detail

// external API

std::unique_ptr<column> encode(column_view const& input_column,
                               data_type indices_type,
                               rmm::mr::device_memory_resource* mr)
{
  CUDF_FUNC_RANGE();
  return detail::encode(input_column, indices_type, mr);
}

}  // namespace dictionary
}  // namespace cudf
