#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2021-2022, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <cudf/column/column_factories.hpp>
#include <cudf/detail/iterator.cuh>
#include <cudf/detail/valid_if.cuh>
#include <cudf/lists/detail/contains.hpp>
#include <cudf/lists/list_device_view.cuh>
#include <cudf/lists/lists_column_device_view.cuh>
#include <cudf/lists/lists_column_view.hpp>
#include <cudf/scalar/scalar.hpp>
#include <cudf/table/experimental/row_operators.cuh>
#include <cudf/table/row_operators.cuh>
#include <cudf/utilities/default_stream.hpp>
#include <cudf/utilities/type_dispatcher.hpp>

#include <rmm/exec_policy.hpp>

#include <thrust/execution_policy.h>
#include <thrust/find.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/iterator/reverse_iterator.h>
#include <thrust/logical.h>
#include <thrust/optional.h>
#include <thrust/pair.h>
#include <thrust/tabulate.h>
#include <thrust/transform.h>

#include <type_traits>

namespace cudf::lists {
namespace {

/**
 * @brief A sentinel value used for marking that a given key has not been found in the search list.
 *
 * The value should be `-1` as indicated in the public API documentation.
 */
auto constexpr __device__ NOT_FOUND_SENTINEL = size_type{-1};

/**
 * @brief A sentinel value used for marking that a given output row should be null.
 *
 * This value should be different from `NOT_FOUND_SENTINEL`.
 */
auto constexpr __device__ NULL_SENTINEL = std::numeric_limits<size_type>::min();

/**
 * @brief Check if the given type is a supported non-nested type in `cudf::lists::contains`.
 */
template <typename Element>
static auto constexpr is_supported_non_nested_type()
{
  return cudf::is_fixed_width<Element>() || std::is_same_v<Element, cudf::string_view>;
}

/**
 * @brief Check if the given type is supported in `cudf::lists::contains`.
 */
template <typename Element>
auto constexpr is_supported_type()
{
  return is_supported_non_nested_type<Element>() || cudf::is_nested<Element>();
}

/**
 * @brief Return a pair of index iterators {begin, end} to loop through elements within a
 * list.
 *
 * Depending on the value of `forward`, a pair of forward or reverse iterators will be
 * returned, allowing to loop through elements in the list in first-to-last or last-to-first
 * order.
 *
 * Note that the element indices always restart to `0` at the first position in each list.
 *
 * @tparam forward A boolean value indicating whether we want to iterate elements in the list
 *         by forward or reverse order.
 * @param size The number of elements in the list.
 * @return A pair of {begin, end} iterators to iterate through the range `[0, size)`.
 */
template <bool forward>
__device__ auto element_index_pair_iter(size_type const size)
{
  auto const begin = thrust::make_counting_iterator(0);
  auto const end   = thrust::make_counting_iterator(size);

  if constexpr (forward) {
    return thrust::pair{begin, end};
  } else {
    return thrust::pair{thrust::make_reverse_iterator(end), thrust::make_reverse_iterator(begin)};
  }
}

/**
 * @brief Functor to perform searching for index of a key element in a given list, specialized
 * for non-nested types.
 */
struct search_list_non_nested_types_fn {
  duplicate_find_option const find_option;

  template <typename Element, CUDF_ENABLE_IF(is_supported_non_nested_type<Element>())>
  __device__ size_type operator()(list_device_view const list,
                                  thrust::optional<Element> const key_opt) const
  {
    // A null list or null key will result in a null output row.
    if (list.is_null() || !key_opt) { return NULL_SENTINEL; }

    return find_option == duplicate_find_option::FIND_FIRST
             ? search_list<true, Element>(list, *key_opt)
             : search_list<false, Element>(list, *key_opt);
  }

  template <typename Element, CUDF_ENABLE_IF(!is_supported_non_nested_type<Element>())>
  __device__ size_type operator()(list_device_view const, thrust::optional<Element> const) const
  {
    CUDF_UNREACHABLE("Unsupported type.");
  }

 private:
  template <bool forward, typename Element, CUDF_ENABLE_IF(is_supported_non_nested_type<Element>())>
  static __device__ inline size_type search_list(list_device_view const list,
                                                 Element const search_key)
  {
    auto const [begin, end] = element_index_pair_iter<forward>(list.size());
    auto const found_iter =
      thrust::find_if(thrust::seq, begin, end, [=] __device__(auto const idx) {
        return !list.is_null(idx) &&
               cudf::equality_compare(list.template element<Element>(idx), search_key);
      });
    // If the key is found, return its found position in the list from `found_iter`.
    return found_iter == end ? NOT_FOUND_SENTINEL : *found_iter;
  }
};

/**
 * @brief Functor to perform searching for index of a key element in a given list, specialized
 * for nested types.
 */
template <typename KeyValidityIter, typename EqComparator>
struct search_list_nested_types_fn {
  duplicate_find_option const find_option;
  KeyValidityIter const key_validity_iter;
  EqComparator const d_comp;
  bool const search_key_is_scalar;

  search_list_nested_types_fn(duplicate_find_option const find_option,
                              KeyValidityIter const key_validity_iter,
                              EqComparator const& d_comp,
                              bool search_key_is_scalar)
    : find_option(find_option),
      key_validity_iter(key_validity_iter),
      d_comp(d_comp),
      search_key_is_scalar(search_key_is_scalar)
  {
  }

  __device__ size_type operator()(list_device_view const list) const
  {
    // A null list or null key will result in a null output row.
    if (list.is_null() || !key_validity_iter[list.row_index()]) { return NULL_SENTINEL; }

    return find_option == duplicate_find_option::FIND_FIRST ? search_list<true>(list)
                                                            : search_list<false>(list);
  }

 private:
  template <bool forward>
  __device__ inline size_type search_list(list_device_view const list) const
  {
    using cudf::experimental::row::lhs_index_type;
    using cudf::experimental::row::rhs_index_type;

    auto const [begin, end] = element_index_pair_iter<forward>(list.size());
    auto const found_iter =
      thrust::find_if(thrust::seq, begin, end, [=] __device__(auto const idx) {
        return !list.is_null(idx) &&
               d_comp(static_cast<lhs_index_type>(list.element_offset(idx)),
                      static_cast<rhs_index_type>(search_key_is_scalar ? 0 : list.row_index()));
      });
    // If the key is found, return its found position in the list from `found_iter`.
    return found_iter == end ? NOT_FOUND_SENTINEL : *found_iter;
  }
};

/**
 * @brief Function to search for key element(s) in the corresponding rows of a lists column,
 * specialized for non-nested types.
 */
template <bool search_key_is_scalar,
          typename Element,
          typename InputIterator,
          typename OutputIterator,
          typename SearchKeyType>
void index_of_non_nested_types(InputIterator input_it,
                               size_type num_rows,
                               OutputIterator output_it,
                               SearchKeyType const& search_keys,
                               bool search_keys_have_nulls,
                               duplicate_find_option find_option,
                               rmm::cuda_stream_view stream)
{
  auto const do_search = [=](auto const keys_iter) {
    thrust::transform(rmm::exec_policy(stream),
                      input_it,
                      input_it + num_rows,
                      keys_iter,
                      output_it,
                      search_list_non_nested_types_fn{find_option});
  };

  if constexpr (search_key_is_scalar) {
    auto const keys_iter = cudf::detail::make_optional_iterator<Element>(
      search_keys, nullate::DYNAMIC{search_keys_have_nulls});
    do_search(keys_iter);
  } else {
    auto const keys_cdv_ptr = column_device_view::create(search_keys, stream);
    auto const keys_iter    = cudf::detail::make_optional_iterator<Element>(
      *keys_cdv_ptr, nullate::DYNAMIC{search_keys_have_nulls});
    do_search(keys_iter);
  }
}

/**
 * @brief Function to search for index of key element(s) in the corresponding rows of a lists
 * column, specialized for nested types.
 */
template <bool search_key_is_scalar,
          typename InputIterator,
          typename OutputIterator,
          typename SearchKeyType>
void index_of_nested_types(InputIterator input_it,
                           size_type num_rows,
                           OutputIterator output_it,
                           column_view const& child,
                           SearchKeyType const& search_keys,
                           duplicate_find_option find_option,
                           rmm::cuda_stream_view stream)
{
  // Create a `table_view` from the search key(s).
  // If the input search key is a (nested type) scalar, a new column is materialized from that
  // scalar before a `table_view` is generated from it. As such, the new created column will also be
  // returned to keep the result `table_view` valid.
  [[maybe_unused]] auto const [keys_tview, unused_column] =
    [&]() -> std::pair<table_view, std::unique_ptr<column>> {
    if constexpr (search_key_is_scalar) {
      auto tmp_column = make_column_from_scalar(search_keys, 1, stream);
      return {table_view{{tmp_column->view()}}, std::move(tmp_column)};
    } else {
      return {table_view{{search_keys}}, nullptr};
    }
  }();

  auto const child_tview = table_view{{child}};
  auto const has_nulls   = has_nested_nulls(child_tview) || has_nested_nulls(keys_tview);
  auto const comparator =
    cudf::experimental::row::equality::two_table_comparator(child_tview, keys_tview, stream);
  auto const d_comp = comparator.equal_to(nullate::DYNAMIC{has_nulls});

  auto const do_search = [=](auto const key_validity_iter) {
    thrust::transform(
      rmm::exec_policy(stream),
      input_it,
      input_it + num_rows,
      output_it,
      search_list_nested_types_fn{find_option, key_validity_iter, d_comp, search_key_is_scalar});
  };

  if constexpr (search_key_is_scalar) {
    auto const key_validity_iter = cudf::detail::make_validity_iterator<true>(search_keys);
    do_search(key_validity_iter);
  } else {
    auto const keys_dv_ptr       = column_device_view::create(search_keys, stream);
    auto const key_validity_iter = cudf::detail::make_validity_iterator<true>(*keys_dv_ptr);
    do_search(key_validity_iter);
  }
}

/**
 * @brief Dispatch functor to search for index of key element(s) in the corresponding rows of a
 * lists column.
 */
struct dispatch_index_of {
  // SFINAE with conditional return type because we need to support device lambda in this function.
  // This is required due to a limitation of nvcc.
  template <typename Element, typename SearchKeyType>
  std::enable_if_t<is_supported_type<Element>(), std::unique_ptr<column>> operator()(
    lists_column_view const& lists,
    SearchKeyType const& search_keys,
    duplicate_find_option find_option,
    rmm::cuda_stream_view stream,
    rmm::mr::device_memory_resource* mr) const
  {
    // Access the child column through `child()` method, not `get_sliced_child()`.
    // This is because slicing offset has already been taken into account during row
    // comparisons.
    auto const child = lists.child();

    CUDF_EXPECTS(child.type() == search_keys.type(),
                 "Type/Scale of search key does not match list column element type.");
    CUDF_EXPECTS(search_keys.type().id() != type_id::EMPTY, "Type cannot be empty.");

    auto constexpr search_key_is_scalar = std::is_same_v<SearchKeyType, cudf::scalar>;
    auto const search_keys_have_nulls   = [&search_keys, stream] {
      if constexpr (search_key_is_scalar) {
        return !search_keys.is_valid(stream);
      } else {
        return search_keys.has_nulls();
      }
    }();

    auto const num_rows = lists.size();

    if (search_key_is_scalar && search_keys_have_nulls) {
      // If the scalar key is invalid/null, the entire output column will be all nulls.
      return make_numeric_column(data_type{cudf::type_to_id<size_type>()},
                                 num_rows,
                                 cudf::create_null_mask(num_rows, mask_state::ALL_NULL, mr),
                                 num_rows,
                                 stream,
                                 mr);
    }

    auto const lists_cdv_ptr = column_device_view::create(lists.parent(), stream);
    auto const input_it      = cudf::detail::make_counting_transform_iterator(
      size_type{0},
      [lists = cudf::detail::lists_column_device_view{*lists_cdv_ptr}] __device__(auto const idx) {
        return list_device_view{lists, idx};
      });

    auto out_positions = make_numeric_column(
      data_type{type_to_id<size_type>()}, num_rows, cudf::mask_state::UNALLOCATED, stream, mr);
    auto const output_it = out_positions->mutable_view().template begin<size_type>();

    if constexpr (not cudf::is_nested<Element>()) {
      index_of_non_nested_types<search_key_is_scalar, Element>(
        input_it, num_rows, output_it, search_keys, search_keys_have_nulls, find_option, stream);
    } else {  // list + struct
      index_of_nested_types<search_key_is_scalar>(
        input_it, num_rows, output_it, child, search_keys, find_option, stream);
    }

    if (search_keys_have_nulls || lists.has_nulls()) {
      auto [null_mask, null_count] = cudf::detail::valid_if(
        output_it,
        output_it + num_rows,
        [] __device__(auto const idx) { return idx != NULL_SENTINEL; },
        stream,
        mr);
      out_positions->set_null_mask(std::move(null_mask), null_count);
    }
    return out_positions;
  }

  template <typename Element, typename SearchKeyType>
  std::enable_if_t<!is_supported_type<Element>(), std::unique_ptr<column>> operator()(
    lists_column_view const&,
    SearchKeyType const&,
    duplicate_find_option,
    rmm::cuda_stream_view,
    rmm::mr::device_memory_resource*) const
  {
    CUDF_FAIL("Unsupported type in `dispatch_index_of` functor.");
  }
};

/**
 * @brief Converts key-positions vector (from `index_of()`) to a BOOL8 vector, indicating if
 * the search key(s) were found.
 */
std::unique_ptr<column> to_contains(std::unique_ptr<column>&& key_positions,
                                    rmm::cuda_stream_view stream,
                                    rmm::mr::device_memory_resource* mr)
{
  CUDF_EXPECTS(key_positions->type().id() == type_to_id<size_type>(),
               "Expected input column of type cudf::size_type.");
  auto const positions_begin = key_positions->view().template begin<size_type>();
  auto result                = make_numeric_column(
    data_type{type_id::BOOL8}, key_positions->size(), mask_state::UNALLOCATED, stream, mr);
  thrust::transform(rmm::exec_policy(stream),
                    positions_begin,
                    positions_begin + key_positions->size(),
                    result->mutable_view().template begin<bool>(),
                    [] __device__(auto const i) {
                      // position == NOT_FOUND_SENTINEL: the list does not contain the search key.
                      return i != NOT_FOUND_SENTINEL;
                    });

  auto const null_count                             = key_positions->null_count();
  [[maybe_unused]] auto [data, null_mask, children] = key_positions->release();
  result->set_null_mask(std::move(*null_mask.release()), null_count);

  return result;
}
}  // namespace

namespace detail {
std::unique_ptr<column> index_of(lists_column_view const& lists,
                                 cudf::scalar const& search_key,
                                 duplicate_find_option find_option,
                                 rmm::cuda_stream_view stream,
                                 rmm::mr::device_memory_resource* mr)
{
  return cudf::type_dispatcher(
    search_key.type(), dispatch_index_of{}, lists, search_key, find_option, stream, mr);
}

std::unique_ptr<column> index_of(lists_column_view const& lists,
                                 column_view const& search_keys,
                                 duplicate_find_option find_option,
                                 rmm::cuda_stream_view stream,
                                 rmm::mr::device_memory_resource* mr)
{
  CUDF_EXPECTS(search_keys.size() == lists.size(),
               "Number of search keys must match list column size.");
  return cudf::type_dispatcher(
    search_keys.type(), dispatch_index_of{}, lists, search_keys, find_option, stream, mr);
}

std::unique_ptr<column> contains(lists_column_view const& lists,
                                 cudf::scalar const& search_key,
                                 rmm::cuda_stream_view stream,
                                 rmm::mr::device_memory_resource* mr)
{
  auto key_indices = index_of(lists,
                              search_key,
                              duplicate_find_option::FIND_FIRST,
                              stream,
                              rmm::mr::get_current_device_resource());
  return to_contains(std::move(key_indices), stream, mr);
}

std::unique_ptr<column> contains(lists_column_view const& lists,
                                 column_view const& search_keys,
                                 rmm::cuda_stream_view stream,
                                 rmm::mr::device_memory_resource* mr)
{
  CUDF_EXPECTS(search_keys.size() == lists.size(),
               "Number of search keys must match list column size.");

  auto key_indices = index_of(lists,
                              search_keys,
                              duplicate_find_option::FIND_FIRST,
                              stream,
                              rmm::mr::get_current_device_resource());
  return to_contains(std::move(key_indices), stream, mr);
}

std::unique_ptr<column> contains_nulls(lists_column_view const& lists,
                                       rmm::cuda_stream_view stream,
                                       rmm::mr::device_memory_resource* mr)
{
  auto const lists_cv      = lists.parent();
  auto output              = make_numeric_column(data_type{type_to_id<bool>()},
                                    lists.size(),
                                    copy_bitmask(lists_cv, stream, mr),
                                    lists_cv.null_count(),
                                    stream,
                                    mr);
  auto const out_begin     = output->mutable_view().template begin<bool>();
  auto const lists_cdv_ptr = column_device_view::create(lists_cv, stream);

  thrust::tabulate(rmm::exec_policy(stream),
                   out_begin,
                   out_begin + lists.size(),
                   [lists = cudf::detail::lists_column_device_view{*lists_cdv_ptr}] __device__(
                     auto const list_idx) {
                     auto const list = list_device_view{lists, list_idx};
                     return list.is_null() ||
                            thrust::any_of(thrust::seq,
                                           thrust::make_counting_iterator(0),
                                           thrust::make_counting_iterator(list.size()),
                                           [&list](auto const idx) { return list.is_null(idx); });
                   });

  return output;
}

}  // namespace detail

std::unique_ptr<column> contains(lists_column_view const& lists,
                                 cudf::scalar const& search_key,
                                 rmm::mr::device_memory_resource* mr)
{
  CUDF_FUNC_RANGE();
  return detail::contains(lists, search_key, cudf::get_default_stream(), mr);
}

std::unique_ptr<column> contains(lists_column_view const& lists,
                                 column_view const& search_keys,
                                 rmm::mr::device_memory_resource* mr)
{
  CUDF_FUNC_RANGE();
  return detail::contains(lists, search_keys, cudf::get_default_stream(), mr);
}

std::unique_ptr<column> contains_nulls(lists_column_view const& lists,
                                       rmm::mr::device_memory_resource* mr)
{
  CUDF_FUNC_RANGE();
  return detail::contains_nulls(lists, cudf::get_default_stream(), mr);
}

std::unique_ptr<column> index_of(lists_column_view const& lists,
                                 cudf::scalar const& search_key,
                                 duplicate_find_option find_option,
                                 rmm::mr::device_memory_resource* mr)
{
  CUDF_FUNC_RANGE();
  return detail::index_of(lists, search_key, find_option, cudf::get_default_stream(), mr);
}

std::unique_ptr<column> index_of(lists_column_view const& lists,
                                 column_view const& search_keys,
                                 duplicate_find_option find_option,
                                 rmm::mr::device_memory_resource* mr)
{
  CUDF_FUNC_RANGE();
  return detail::index_of(lists, search_keys, find_option, cudf::get_default_stream(), mr);
}

}  // namespace cudf::lists
