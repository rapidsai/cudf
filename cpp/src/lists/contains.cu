#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2021-2022, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <cudf_test/column_utilities.hpp>

#include <cudf/column/column_factories.hpp>
#include <cudf/detail/iterator.cuh>
#include <cudf/detail/valid_if.cuh>
#include <cudf/lists/detail/contains.hpp>
#include <cudf/lists/list_device_view.cuh>
#include <cudf/lists/lists_column_device_view.cuh>
#include <cudf/lists/lists_column_view.hpp>
#include <cudf/scalar/scalar.hpp>
#include <cudf/table/row_operators.cuh>
#include <cudf/utilities/type_dispatcher.hpp>

#include <rmm/exec_policy.hpp>

#include <thrust/execution_policy.h>
#include <thrust/find.h>
#include <thrust/functional.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/iterator/reverse_iterator.h>
#include <thrust/logical.h>
#include <thrust/tabulate.h>
#include <thrust/transform.h>

#include <type_traits>

namespace cudf::lists {

namespace {

/**
 * @brief A sentinel value used for marking that a given key has not been found in the search list.
 *
 * The value should be `-1` as indicated in the public API documentation.
 */
auto constexpr __device__ NOT_FOUND_SENTINEL = size_type{-1};

/**
 * @brief A sentinel value used for marking that a given output row should be null.
 */
auto constexpr __device__ NULL_SENTINEL = std::numeric_limits<size_type>::min();

/**
 * @brief The functor to identify if an output row is valid.
 */
struct is_valid_fn {
  __device__ bool operator()(size_type const idx) const noexcept
  {
    printf("%d\n", idx);
    return idx != NULL_SENTINEL;
  }
};

template <typename Element>
auto constexpr is_supported_non_nested_type()
{
  return cudf::is_fixed_width<Element>() || std::is_same_v<Element, cudf::string_view>;
}

/**
 * @brief Return a pair of index iterators {begin, end} to loop through elements within a list.
 *
 * Depending on the value of `find_first`, a pair of forward or reverse iterators will be returned,
 * allowing to loop through elements in the list in first-to-last or last-to-first order.
 *
 * Note that the element indices always restart to `0` at the first position in each list.
 *
 * @tparam find_first A boolean value indicating whether we want to find the first or last
 *         appearance of a given key in the list.
 * @param size The number of elements in the list.
 * @return A pair of {begin, end} iterators to iterate through the range `[0, size)` by forward or
 *         reverse order.
 */
template <bool find_first>
auto __device__ element_index_pair_iter(size_type const size)
{
  if constexpr (find_first) {
    return thrust::pair(thrust::make_counting_iterator(0), thrust::make_counting_iterator(size));
  } else {
    return thrust::pair(thrust::make_reverse_iterator(thrust::make_counting_iterator(size)),
                        thrust::make_reverse_iterator(thrust::make_counting_iterator(0)));
  }
}

template <typename SearchKeyIter>
struct search_lists_fn {
  cudf::detail::lists_column_device_view const lists;
  SearchKeyIter const keys_iter;
  duplicate_find_option const find_option;

  search_lists_fn(cudf::detail::lists_column_device_view const lists,
                  SearchKeyIter const keys_iter,
                  duplicate_find_option const find_option)
    : lists{lists}, keys_iter{keys_iter}, find_option{find_option}
  {
  }

  __device__ size_type operator()(size_type const idx) const
  {
    auto const list = list_device_view{lists, idx};
    // A null list never contains any key, even null key.
    // In addition, a null list will result in a null output row.
    if (list.is_null()) { return NULL_SENTINEL; }

    auto const key_opt = keys_iter[idx];
    // A null key will also result in a null output row.
    if (!key_opt) { return NULL_SENTINEL; }

    auto const& key = key_opt.value();

    using Element = typename thrust::iterator_traits<SearchKeyIter>::value_type::value_type;
    return find_option == duplicate_find_option::FIND_FIRST
             ? search_list<Element, true>(list, key)
             : search_list<Element, false>(list, key);
  }

 private:
  template <typename Element, bool find_first>
  static __device__ size_type search_list(list_device_view const& list, Element const& search_key)
  {
    auto const [begin, end] = element_index_pair_iter<find_first>(list.size());
    auto const found_iter =
      thrust::find_if(thrust::seq, begin, end, [&] __device__(auto const idx) {
        return !list.is_null(idx) &&
               cudf::equality_compare(list.template element<Element>(idx), search_key);
      });
    // If the key is found, return its found position in the list from `found_iter`.
    return found_iter == end ? NOT_FOUND_SENTINEL : *found_iter;
  }
};

/**
 * @brief Dispatch functor to search for key element(s) in the corresponding rows of a lists column.
 */
struct dispatch_index_of {
  template <typename Element,
            typename SearchKeyType,
            CUDF_ENABLE_IF(is_supported_non_nested_type<Element>())>
  std::unique_ptr<column> operator()(lists_column_view const& lists,
                                     SearchKeyType const& search_keys,
                                     duplicate_find_option find_option,
                                     rmm::cuda_stream_view stream,
                                     rmm::mr::device_memory_resource* mr) const
  {
    CUDF_EXPECTS(!cudf::is_nested(lists.child().type()),
                 "Nested types not supported in list search operations.");
    CUDF_EXPECTS(lists.child().type() == search_keys.type(),
                 "Type/Scale of search key does not match list column element type.");
    CUDF_EXPECTS(search_keys.type().id() != type_id::EMPTY, "Type cannot be empty.");

    auto constexpr search_key_is_scalar = std::is_same_v<SearchKeyType, cudf::scalar>;
    auto const search_keys_have_nulls   = [&] {
      if constexpr (search_key_is_scalar) {
        return !search_keys.is_valid(stream);
      } else {
        return search_keys.has_nulls();
      }
    }();

    if (search_key_is_scalar && search_keys_have_nulls) {
      // If the scalar key is invalid/null, the entire output column will be all nulls.
      return make_numeric_column(data_type{cudf::type_to_id<size_type>()},
                                 lists.size(),
                                 cudf::create_null_mask(lists.size(), mask_state::ALL_NULL, mr),
                                 lists.size(),
                                 stream,
                                 mr);
    }

    auto const lists_cdv_ptr = column_device_view::create(lists.parent(), stream);
    auto const lists_cdv     = cudf::detail::lists_column_device_view{*lists_cdv_ptr};

    auto out_positions = make_numeric_column(
      data_type{type_to_id<size_type>()}, lists.size(), cudf::mask_state::UNALLOCATED, stream, mr);
    auto const out_begin = out_positions->mutable_view().template begin<size_type>();

    auto const do_search = [&](auto const keys_iter) {
      thrust::tabulate(rmm::exec_policy(stream),
                       out_begin,
                       out_begin + lists.size(),
                       search_lists_fn{lists_cdv, keys_iter, find_option});
    };

    if constexpr (search_key_is_scalar) {
      auto const keys_iter = cudf::detail::make_optional_iterator<Element>(
        search_keys, nullate::DYNAMIC{search_keys_have_nulls});
      do_search(keys_iter);
    } else {
      auto const keys_cdv_ptr = column_device_view::create(search_keys, stream);
      auto const keys_iter    = cudf::detail::make_optional_iterator<Element>(
        *keys_cdv_ptr, nullate::DYNAMIC{search_keys_have_nulls});
      do_search(keys_iter);
    }

    cudf::test::print(out_positions->view());

    if (search_keys_have_nulls || lists.has_nulls()) {
      auto [null_mask, null_count] =
        cudf::detail::valid_if(out_begin, out_begin + lists.size(), is_valid_fn{}, stream, mr);
      out_positions->set_null_mask(std::move(null_mask), null_count);
    }
    return out_positions;
  }

  template <typename Element,
            typename SearchKeyType,
            CUDF_ENABLE_IF(!is_supported_non_nested_type<Element>())>
  std::unique_ptr<column> operator()(lists_column_view const&,
                                     SearchKeyType const&,
                                     duplicate_find_option,
                                     rmm::cuda_stream_view,
                                     rmm::mr::device_memory_resource*) const
  {
    CUDF_FAIL("Unsupported type in `dispatch_index_of` functor.");
  }
};

/**
 * @brief Converts key-positions vector (from `index_of()`) to a BOOL8 vector, indicating if
 *        the search key(s) were found.
 */
std::unique_ptr<column> to_contains(std::unique_ptr<column>&& key_positions,
                                    rmm::cuda_stream_view stream,
                                    rmm::mr::device_memory_resource* mr)
{
  CUDF_EXPECTS(key_positions->type().id() == type_to_id<size_type>(),
               "Expected input column of type cudf::size_type.");
  auto const positions_begin = key_positions->view().template begin<size_type>();
  auto result                = make_numeric_column(
    data_type{type_id::BOOL8}, key_positions->size(), mask_state::UNALLOCATED, stream, mr);
  thrust::transform(rmm::exec_policy(stream),
                    positions_begin,
                    positions_begin + key_positions->size(),
                    result->mutable_view().template begin<bool>(),
                    [] __device__(auto const i) {
                      // position == NOT_FOUND_SENTINEL: the list does not contain the search key.
                      return i != NOT_FOUND_SENTINEL;
                    });

  auto const null_count                             = key_positions->null_count();
  [[maybe_unused]] auto [data, null_mask, children] = key_positions->release();
  result->set_null_mask(std::move(*null_mask.release()), null_count);

  return result;
}
}  // namespace

namespace detail {
std::unique_ptr<column> index_of(lists_column_view const& lists,
                                 cudf::scalar const& search_key,
                                 duplicate_find_option find_option,
                                 rmm::cuda_stream_view stream,
                                 rmm::mr::device_memory_resource* mr)
{
  return cudf::type_dispatcher(
    search_key.type(), dispatch_index_of{}, lists, search_key, find_option, stream, mr);
}

std::unique_ptr<column> index_of(lists_column_view const& lists,
                                 column_view const& search_keys,
                                 duplicate_find_option find_option,
                                 rmm::cuda_stream_view stream,
                                 rmm::mr::device_memory_resource* mr)
{
  CUDF_EXPECTS(search_keys.size() == lists.size(),
               "Number of search keys must match list column size.");
  return cudf::type_dispatcher(
    search_keys.type(), dispatch_index_of{}, lists, search_keys, find_option, stream, mr);
}

std::unique_ptr<column> contains(lists_column_view const& lists,
                                 cudf::scalar const& search_key,
                                 rmm::cuda_stream_view stream,
                                 rmm::mr::device_memory_resource* mr)
{
  return to_contains(
    index_of(lists, search_key, duplicate_find_option::FIND_FIRST, stream), stream, mr);
}

std::unique_ptr<column> contains(lists_column_view const& lists,
                                 column_view const& search_keys,
                                 rmm::cuda_stream_view stream,
                                 rmm::mr::device_memory_resource* mr)
{
  CUDF_EXPECTS(search_keys.size() == lists.size(),
               "Number of search keys must match list column size.");

  return to_contains(
    index_of(lists, search_keys, duplicate_find_option::FIND_FIRST, stream), stream, mr);
}

std::unique_ptr<column> contains_nulls(lists_column_view const& lists,
                                       rmm::cuda_stream_view stream,
                                       rmm::mr::device_memory_resource* mr)
{
  auto const lists_cv      = lists.parent();
  auto output              = make_numeric_column(data_type{type_to_id<bool>()},
                                    lists.size(),
                                    copy_bitmask(lists_cv),
                                    lists_cv.null_count(),
                                    stream,
                                    mr);
  auto const out_begin     = output->mutable_view().template begin<bool>();
  auto const lists_cdv_ptr = column_device_view::create(lists_cv, stream);

  thrust::tabulate(rmm::exec_policy(stream),
                   out_begin,
                   out_begin + lists.size(),
                   [lists = cudf::detail::lists_column_device_view{*lists_cdv_ptr}] __device__(
                     auto const list_idx) {
                     auto const list = list_device_view{lists, list_idx};
                     return list.is_null() ||
                            thrust::any_of(thrust::seq,
                                           thrust::make_counting_iterator(0),
                                           thrust::make_counting_iterator(list.size()),
                                           [&list](auto const idx) { return list.is_null(idx); });
                   });

  return output;
}

}  // namespace detail

std::unique_ptr<column> contains(lists_column_view const& lists,
                                 cudf::scalar const& search_key,
                                 rmm::mr::device_memory_resource* mr)
{
  CUDF_FUNC_RANGE();
  return detail::contains(lists, search_key, rmm::cuda_stream_default, mr);
}

std::unique_ptr<column> contains(lists_column_view const& lists,
                                 column_view const& search_keys,
                                 rmm::mr::device_memory_resource* mr)
{
  CUDF_FUNC_RANGE();
  return detail::contains(lists, search_keys, rmm::cuda_stream_default, mr);
}

std::unique_ptr<column> contains_nulls(lists_column_view const& lists,
                                       rmm::mr::device_memory_resource* mr)
{
  CUDF_FUNC_RANGE();
  return detail::contains_nulls(lists, rmm::cuda_stream_default, mr);
}

std::unique_ptr<column> index_of(lists_column_view const& lists,
                                 cudf::scalar const& search_key,
                                 duplicate_find_option find_option,
                                 rmm::mr::device_memory_resource* mr)
{
  CUDF_FUNC_RANGE();
  return detail::index_of(lists, search_key, find_option, rmm::cuda_stream_default, mr);
}

std::unique_ptr<column> index_of(lists_column_view const& lists,
                                 column_view const& search_keys,
                                 duplicate_find_option find_option,
                                 rmm::mr::device_memory_resource* mr)
{
  CUDF_FUNC_RANGE();
  return detail::index_of(lists, search_keys, find_option, rmm::cuda_stream_default, mr);
}

}  // namespace cudf::lists
