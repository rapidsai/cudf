#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2021-2023, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <cudf/column/column_factories.hpp>
#include <cudf/detail/iterator.cuh>
#include <cudf/detail/valid_if.cuh>
#include <cudf/lists/detail/contains.hpp>
#include <cudf/lists/detail/lists_column_factories.hpp>
#include <cudf/lists/list_device_view.cuh>
#include <cudf/lists/lists_column_device_view.cuh>
#include <cudf/lists/lists_column_view.hpp>
#include <cudf/scalar/scalar.hpp>
#include <cudf/table/experimental/row_operators.cuh>
#include <cudf/table/row_operators.cuh>
#include <cudf/utilities/default_stream.hpp>
#include <cudf/utilities/type_dispatcher.hpp>

#include <rmm/exec_policy.hpp>

#include <thrust/execution_policy.h>
#include <thrust/find.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/iterator/reverse_iterator.h>
#include <thrust/logical.h>
#include <thrust/optional.h>
#include <thrust/pair.h>
#include <thrust/tabulate.h>
#include <thrust/transform.h>

#include <type_traits>

namespace cudf::lists {
namespace {

/**
 * @brief A sentinel value used for marking that a given key has not been found in the search list.
 *
 * The value should be `-1` as indicated in the public API documentation.
 */
auto constexpr __device__ NOT_FOUND_SENTINEL = size_type{-1};

/**
 * @brief A sentinel value used for marking that a given output row should be null.
 *
 * This value should be different from `NOT_FOUND_SENTINEL`.
 */
auto constexpr __device__ NULL_SENTINEL = std::numeric_limits<size_type>::min();

/**
 * @brief Check if the given type is a supported non-nested type in `cudf::lists::contains`.
 */
template <typename Element>
static auto constexpr is_supported_non_nested_type()
{
  return cudf::is_fixed_width<Element>() || std::is_same_v<Element, cudf::string_view>;
}

/**
 * @brief Check if the given type is supported in `cudf::lists::contains`.
 */
struct is_supported_type_fn {
  template <typename Element>
  auto constexpr operator()()
  {
    return is_supported_non_nested_type<Element>() || cudf::is_nested<Element>();
  }
};

/**
 * @brief Return a pair of index iterators {begin, end} to loop through elements within a
 * list.
 *
 * Depending on the value of `forward`, a pair of forward or reverse iterators will be
 * returned, allowing to loop through elements in the list in first-to-last or last-to-first
 * order.
 *
 * Note that the element indices always restart to `0` at the first position in each list.
 *
 * @tparam forward A boolean value indicating whether we want to iterate elements in the list
 *         by forward or reverse order.
 * @param size The number of elements in the list.
 * @return A pair of {begin, end} iterators to iterate through the range `[0, size)`.
 */
template <bool forward>
__device__ auto element_index_pair_iter(size_type const size)
{
  auto const begin = thrust::make_counting_iterator(0);
  auto const end   = thrust::make_counting_iterator(size);

  if constexpr (forward) {
    return thrust::pair{begin, end};
  } else {
    return thrust::pair{thrust::make_reverse_iterator(end), thrust::make_reverse_iterator(begin)};
  }
}

/**
 * @brief Functor to perform searching for index of a key element in a given list, specialized
 * for nested types.
 */
template <typename KeyValidityIter, typename EqComparator>
struct search_list_fn {
  duplicate_find_option const find_option;
  KeyValidityIter const key_validity_iter;
  EqComparator const d_comp;

  search_list_fn(duplicate_find_option const find_option,
                 KeyValidityIter const key_validity_iter,
                 EqComparator const& d_comp)
    : find_option(find_option), key_validity_iter(key_validity_iter), d_comp(d_comp)
  {
  }

  __device__ size_type operator()(list_device_view const list) const
  {
    // A null list or null key will result in a null output row.
    if (list.is_null() || !key_validity_iter[list.row_index()]) { return NULL_SENTINEL; }

    return find_option == duplicate_find_option::FIND_FIRST ? search_list_op<true>(list)
                                                            : search_list_op<false>(list);
  }

 private:
  template <bool forward>
  __device__ inline size_type search_list_op(list_device_view const list) const
  {
    using cudf::experimental::row::lhs_index_type;
    using cudf::experimental::row::rhs_index_type;

    auto const [begin, end] = element_index_pair_iter<forward>(list.size());
    auto const found_iter =
      thrust::find_if(thrust::seq, begin, end, [=] __device__(auto const idx) {
        return !list.is_null(idx) && d_comp(static_cast<lhs_index_type>(list.element_offset(idx)),
                                            static_cast<rhs_index_type>(list.row_index()));
      });
    // If the key is found, return its found position in the list from `found_iter`.
    return found_iter == end ? NOT_FOUND_SENTINEL : *found_iter;
  }
};

/**
 * @brief Function to search for index of key element(s) in the corresponding rows of a lists
 * column, specialized for nested types.
 */
template <typename InputIterator, typename OutputIterator, typename DeviceComp>
void index_of(InputIterator input_it,
              size_type num_rows,
              OutputIterator output_it,
              column_view const& child,
              column_view const& search_keys,
              duplicate_find_option find_option,
              DeviceComp d_comp,
              rmm::cuda_stream_view stream)
{
  auto const keys_dv_ptr       = column_device_view::create(search_keys, stream);
  auto const key_validity_iter = cudf::detail::make_validity_iterator<true>(*keys_dv_ptr);
  thrust::transform(rmm::exec_policy(stream),
                    input_it,
                    input_it + num_rows,
                    output_it,
                    search_list_fn{find_option, key_validity_iter, d_comp});
}

/**
 * @brief Dispatch function to search for index of key element(s) in the corresponding rows of a
 * lists column.
 */
std::unique_ptr<column> dispatch_index_of(lists_column_view const& lists,
                                          column_view const& search_keys,
                                          duplicate_find_option find_option,
                                          rmm::cuda_stream_view stream,
                                          rmm::mr::device_memory_resource* mr)
{
  CUDF_EXPECTS(cudf::type_dispatcher(search_keys.type(), is_supported_type_fn{}),
               "Unsupported type in `dispatch_index_of` function.");
  // Access the child column through `child()` method, not `get_sliced_child()`.
  // This is because slicing offset has already been taken into account during row
  // comparisons.
  auto const child = lists.child();

  CUDF_EXPECTS(child.type() == search_keys.type(),
               "Type/Scale of search key does not match list column element type.",
               cudf::data_type_error);
  CUDF_EXPECTS(search_keys.type().id() != type_id::EMPTY, "Type cannot be empty.");

  auto const search_keys_have_nulls = search_keys.has_nulls();

  auto const num_rows = lists.size();

  auto const lists_cdv_ptr = column_device_view::create(lists.parent(), stream);
  auto const input_it      = cudf::detail::make_counting_transform_iterator(
    size_type{0},
    [lists = cudf::detail::lists_column_device_view{*lists_cdv_ptr}] __device__(auto const idx) {
      return list_device_view{lists, idx};
    });

  auto out_positions = make_numeric_column(
    data_type{type_to_id<size_type>()}, num_rows, cudf::mask_state::UNALLOCATED, stream, mr);
  auto const output_it = out_positions->mutable_view().template begin<size_type>();

  auto const keys_tview  = cudf::table_view{{search_keys}};
  auto const child_tview = cudf::table_view{{child}};
  auto const has_nulls   = has_nested_nulls(child_tview) || has_nested_nulls(keys_tview);
  auto const comparator =
    cudf::experimental::row::equality::two_table_comparator(child_tview, keys_tview, stream);
  if (cudf::is_nested(search_keys.type())) {
    auto const d_comp = comparator.equal_to<true>(nullate::DYNAMIC{has_nulls});
    index_of(input_it, num_rows, output_it, child, search_keys, find_option, d_comp, stream);
  } else {
    auto const d_comp = comparator.equal_to<false>(nullate::DYNAMIC{has_nulls});
    index_of(input_it, num_rows, output_it, child, search_keys, find_option, d_comp, stream);
  }

  if (search_keys_have_nulls || lists.has_nulls()) {
    auto [null_mask, null_count] = cudf::detail::valid_if(
      output_it,
      output_it + num_rows,
      [] __device__(auto const idx) { return idx != NULL_SENTINEL; },
      stream,
      mr);
    out_positions->set_null_mask(std::move(null_mask), null_count);
  }
  return out_positions;
}

/**
 * @brief Converts key-positions vector (from `index_of()`) to a BOOL8 vector, indicating if
 * the search key(s) were found.
 */
std::unique_ptr<column> to_contains(std::unique_ptr<column>&& key_positions,
                                    rmm::cuda_stream_view stream,
                                    rmm::mr::device_memory_resource* mr)
{
  CUDF_EXPECTS(key_positions->type().id() == type_to_id<size_type>(),
               "Expected input column of type cudf::size_type.");
  auto const positions_begin = key_positions->view().template begin<size_type>();
  auto result                = make_numeric_column(
    data_type{type_id::BOOL8}, key_positions->size(), mask_state::UNALLOCATED, stream, mr);
  thrust::transform(rmm::exec_policy(stream),
                    positions_begin,
                    positions_begin + key_positions->size(),
                    result->mutable_view().template begin<bool>(),
                    [] __device__(auto const i) {
                      // position == NOT_FOUND_SENTINEL: the list does not contain the search key.
                      return i != NOT_FOUND_SENTINEL;
                    });

  auto const null_count                             = key_positions->null_count();
  [[maybe_unused]] auto [data, null_mask, children] = key_positions->release();
  result->set_null_mask(std::move(*null_mask.release()), null_count);

  return result;
}
}  // namespace

namespace detail {
std::unique_ptr<column> index_of(lists_column_view const& lists,
                                 cudf::scalar const& search_key,
                                 duplicate_find_option find_option,
                                 rmm::cuda_stream_view stream,
                                 rmm::mr::device_memory_resource* mr)
{
  if (!search_key.is_valid(stream)) {
    return make_numeric_column(data_type{cudf::type_to_id<size_type>()},
                               lists.size(),
                               cudf::create_null_mask(lists.size(), mask_state::ALL_NULL, mr),
                               lists.size(),
                               stream,
                               mr);
  }
  if (lists.size() == 0) {
    return make_numeric_column(
      data_type{type_to_id<size_type>()}, 0, cudf::mask_state::UNALLOCATED, stream, mr);
  }

  auto search_key_col = cudf::make_column_from_scalar(search_key, lists.size(), stream, mr);
  return detail::index_of(lists, search_key_col->view(), find_option, stream, mr);
}

std::unique_ptr<column> index_of(lists_column_view const& lists,
                                 column_view const& search_keys,
                                 duplicate_find_option find_option,
                                 rmm::cuda_stream_view stream,
                                 rmm::mr::device_memory_resource* mr)
{
  CUDF_EXPECTS(search_keys.size() == lists.size(),
               "Number of search keys must match list column size.");
  return dispatch_index_of(lists, search_keys, find_option, stream, mr);
}

std::unique_ptr<column> contains(lists_column_view const& lists,
                                 cudf::scalar const& search_key,
                                 rmm::cuda_stream_view stream,
                                 rmm::mr::device_memory_resource* mr)
{
  auto key_indices = detail::index_of(lists,
                                      search_key,
                                      duplicate_find_option::FIND_FIRST,
                                      stream,
                                      rmm::mr::get_current_device_resource());
  return to_contains(std::move(key_indices), stream, mr);
}

std::unique_ptr<column> contains(lists_column_view const& lists,
                                 column_view const& search_keys,
                                 rmm::cuda_stream_view stream,
                                 rmm::mr::device_memory_resource* mr)
{
  CUDF_EXPECTS(search_keys.size() == lists.size(),
               "Number of search keys must match list column size.");

  auto key_indices = detail::index_of(lists,
                                      search_keys,
                                      duplicate_find_option::FIND_FIRST,
                                      stream,
                                      rmm::mr::get_current_device_resource());
  return to_contains(std::move(key_indices), stream, mr);
}

std::unique_ptr<column> contains_nulls(lists_column_view const& lists,
                                       rmm::cuda_stream_view stream,
                                       rmm::mr::device_memory_resource* mr)
{
  auto const lists_cv      = lists.parent();
  auto output              = make_numeric_column(data_type{type_to_id<bool>()},
                                    lists.size(),
                                    copy_bitmask(lists_cv, stream, mr),
                                    lists_cv.null_count(),
                                    stream,
                                    mr);
  auto const out_begin     = output->mutable_view().template begin<bool>();
  auto const lists_cdv_ptr = column_device_view::create(lists_cv, stream);

  thrust::tabulate(rmm::exec_policy(stream),
                   out_begin,
                   out_begin + lists.size(),
                   [lists = cudf::detail::lists_column_device_view{*lists_cdv_ptr}] __device__(
                     auto const list_idx) {
                     auto const list = list_device_view{lists, list_idx};
                     return list.is_null() ||
                            thrust::any_of(thrust::seq,
                                           thrust::make_counting_iterator(0),
                                           thrust::make_counting_iterator(list.size()),
                                           [&list](auto const idx) { return list.is_null(idx); });
                   });

  return output;
}

}  // namespace detail

std::unique_ptr<column> contains(lists_column_view const& lists,
                                 cudf::scalar const& search_key,
                                 rmm::cuda_stream_view stream,
                                 rmm::mr::device_memory_resource* mr)
{
  CUDF_FUNC_RANGE();
  return detail::contains(lists, search_key, stream, mr);
}

std::unique_ptr<column> contains(lists_column_view const& lists,
                                 column_view const& search_keys,
                                 rmm::cuda_stream_view stream,
                                 rmm::mr::device_memory_resource* mr)
{
  CUDF_FUNC_RANGE();
  return detail::contains(lists, search_keys, stream, mr);
}

std::unique_ptr<column> contains_nulls(lists_column_view const& lists,
                                       rmm::cuda_stream_view stream,
                                       rmm::mr::device_memory_resource* mr)
{
  CUDF_FUNC_RANGE();
  return detail::contains_nulls(lists, stream, mr);
}

std::unique_ptr<column> index_of(lists_column_view const& lists,
                                 cudf::scalar const& search_key,
                                 duplicate_find_option find_option,
                                 rmm::cuda_stream_view stream,
                                 rmm::mr::device_memory_resource* mr)
{
  CUDF_FUNC_RANGE();
  return detail::index_of(lists, search_key, find_option, stream, mr);
}

std::unique_ptr<column> index_of(lists_column_view const& lists,
                                 column_view const& search_keys,
                                 duplicate_find_option find_option,
                                 rmm::cuda_stream_view stream,
                                 rmm::mr::device_memory_resource* mr)
{
  CUDF_FUNC_RANGE();
  return detail::index_of(lists, search_keys, find_option, stream, mr);
}

}  // namespace cudf::lists
