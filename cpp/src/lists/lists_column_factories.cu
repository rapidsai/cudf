/*
 * Copyright (c) 2020-2021, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <cudf/column/column.hpp>
#include <cudf/column/column_factories.hpp>
#include <cudf/column/column_view.hpp>
#include <cudf/detail/gather.cuh>

#include <rmm/cuda_stream_view.hpp>
#include <rmm/exec_policy.hpp>

#include <thrust/iterator/constant_iterator.h>
#include <thrust/sequence.h>

namespace cudf {
namespace lists {
namespace detail {

std::unique_ptr<cudf::column> make_lists_column_from_scalar(list_scalar const& value,
                                                            size_type size,
                                                            rmm::cuda_stream_view stream,
                                                            rmm::mr::device_memory_resource* mr)
{
  if (size == 0) {
    return make_lists_column(0,
                             make_empty_column(data_type{type_to_id<offset_type>()}),
                             empty_like(value.view()),
                             0,
                             cudf::detail::create_null_mask(0, mask_state::UNALLOCATED, stream, mr),
                             stream,
                             mr);
  }
  auto mr_final = size == 1 ? mr : rmm::mr::get_current_device_resource();

  // Handcraft a 1-row column
  auto offsets = make_numeric_column(
    data_type{type_to_id<offset_type>()}, 2, mask_state::UNALLOCATED, stream, mr_final);
  auto m_offsets = offsets->mutable_view();
  thrust::sequence(rmm::exec_policy(stream),
                   m_offsets.begin<size_type>(),
                   m_offsets.end<size_type>(),
                   0,
                   value.view().size());
  size_type null_count = value.is_valid(stream) ? 0 : 1;
  auto null_mask_state = null_count ? mask_state::ALL_NULL : mask_state::UNALLOCATED;
  auto null_mask       = cudf::detail::create_null_mask(1, null_mask_state, stream, mr_final);

  if (size == 1) {
    auto child = std::make_unique<column>(value.view(), stream, mr_final);
    return make_lists_column(
      1, std::move(offsets), std::move(child), null_count, std::move(null_mask), stream, mr_final);
  }

  auto children_views   = std::vector<column_view>{offsets->view(), value.view()};
  auto one_row_col_view = column_view(data_type{type_id::LIST},
                                      1,
                                      nullptr,
                                      static_cast<bitmask_type const*>(null_mask.data()),
                                      null_count,
                                      0,
                                      children_views);

  auto begin = thrust::make_constant_iterator(0);
  auto res   = cudf::detail::gather(table_view({one_row_col_view}),
                                  begin,
                                  begin + size,
                                  out_of_bounds_policy::DONT_CHECK,
                                  stream,
                                  mr_final);
  return std::move(res->release()[0]);
}

}  // namespace detail
}  // namespace lists

/**
 * @copydoc cudf::make_lists_column
 */
std::unique_ptr<column> make_lists_column(size_type num_rows,
                                          std::unique_ptr<column> offsets_column,
                                          std::unique_ptr<column> child_column,
                                          size_type null_count,
                                          rmm::device_buffer&& null_mask,
                                          rmm::cuda_stream_view stream,
                                          rmm::mr::device_memory_resource* mr)
{
  if (null_count > 0) { CUDF_EXPECTS(null_mask.size() > 0, "Column with nulls must be nullable."); }
  CUDF_EXPECTS(
    (num_rows == 0 && offsets_column->size() == 0) || num_rows == offsets_column->size() - 1,
    "Invalid offsets column size for lists column.");
  CUDF_EXPECTS(offsets_column->null_count() == 0, "Offsets column should not contain nulls");
  CUDF_EXPECTS(child_column != nullptr, "Must pass a valid child column");

  std::vector<std::unique_ptr<column>> children;
  children.emplace_back(std::move(offsets_column));
  children.emplace_back(std::move(child_column));
  return std::make_unique<column>(cudf::data_type{type_id::LIST},
                                  num_rows,
                                  rmm::device_buffer{0, stream, mr},
                                  null_mask,
                                  null_count,
                                  std::move(children));
}

}  // namespace cudf
