/*
 * Copyright (c) 2020-2023, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <cudf/column/column.hpp>
#include <cudf/column/column_factories.hpp>
#include <cudf/column/column_view.hpp>
#include <cudf/detail/copy.hpp>
#include <cudf/detail/gather.cuh>
#include <cudf/detail/sizes_to_offsets_iterator.cuh>
#include <cudf/detail/utilities/vector_factories.hpp>
#include <cudf/lists/detail/lists_column_factories.hpp>

#include <rmm/cuda_stream_view.hpp>
#include <rmm/exec_policy.hpp>

#include <thrust/iterator/constant_iterator.h>
#include <thrust/sequence.h>

namespace cudf {
namespace lists {
namespace detail {

std::unique_ptr<cudf::column> make_lists_column_from_scalar(list_scalar const& value,
                                                            size_type size,
                                                            rmm::cuda_stream_view stream,
                                                            rmm::mr::device_memory_resource* mr)
{
  if (size == 0) {
    return make_lists_column(0,
                             make_empty_column(type_to_id<size_type>()),
                             empty_like(value.view()),
                             0,
                             cudf::detail::create_null_mask(0, mask_state::UNALLOCATED, stream, mr),
                             stream,
                             mr);
  }
  auto mr_final = size == 1 ? mr : rmm::mr::get_current_device_resource();

  // Handcraft a 1-row column
  auto sizes_itr = thrust::constant_iterator<size_type>(value.view().size());
  auto offsets   = std::get<0>(
    cudf::detail::make_offsets_child_column(sizes_itr, sizes_itr + 1, stream, mr_final));
  size_type null_count = value.is_valid(stream) ? 0 : 1;
  auto null_mask_state = null_count ? mask_state::ALL_NULL : mask_state::UNALLOCATED;
  auto null_mask       = cudf::detail::create_null_mask(1, null_mask_state, stream, mr_final);

  if (size == 1) {
    auto child = std::make_unique<column>(value.view(), stream, mr_final);
    return make_lists_column(
      1, std::move(offsets), std::move(child), null_count, std::move(null_mask), stream, mr_final);
  }

  auto children_views   = std::vector<column_view>{offsets->view(), value.view()};
  auto one_row_col_view = column_view(data_type{type_id::LIST},
                                      1,
                                      nullptr,
                                      static_cast<bitmask_type const*>(null_mask.data()),
                                      null_count,
                                      0,
                                      children_views);

  auto begin = thrust::make_constant_iterator(0);
  auto res   = cudf::detail::gather(table_view({one_row_col_view}),
                                  begin,
                                  begin + size,
                                  out_of_bounds_policy::DONT_CHECK,
                                  stream,
                                  mr_final);
  return std::move(res->release()[0]);
}

std::unique_ptr<column> make_empty_lists_column(data_type child_type,
                                                rmm::cuda_stream_view stream,
                                                rmm::mr::device_memory_resource* mr)
{
  auto offsets = make_empty_column(data_type(type_to_id<size_type>()));
  auto child   = make_empty_column(child_type);
  return make_lists_column(
    0, std::move(offsets), std::move(child), 0, rmm::device_buffer{}, stream, mr);
}

std::unique_ptr<column> make_all_nulls_lists_column(size_type size,
                                                    data_type child_type,
                                                    rmm::cuda_stream_view stream,
                                                    rmm::mr::device_memory_resource* mr)
{
  auto offsets = [&] {
    auto offsets_buff =
      cudf::detail::make_zeroed_device_uvector_async<size_type>(size + 1, stream, mr);
    return std::make_unique<column>(std::move(offsets_buff), rmm::device_buffer{}, 0);
  }();
  auto child     = make_empty_column(child_type);
  auto null_mask = cudf::detail::create_null_mask(size, mask_state::ALL_NULL, stream, mr);
  return make_lists_column(
    size, std::move(offsets), std::move(child), size, std::move(null_mask), stream, mr);
}

}  // namespace detail
}  // namespace lists

/**
 * @copydoc cudf::make_lists_column
 */
std::unique_ptr<column> make_lists_column(size_type num_rows,
                                          std::unique_ptr<column> offsets_column,
                                          std::unique_ptr<column> child_column,
                                          size_type null_count,
                                          rmm::device_buffer&& null_mask,
                                          rmm::cuda_stream_view stream,
                                          rmm::mr::device_memory_resource* mr)
{
  if (null_count > 0) { CUDF_EXPECTS(null_mask.size() > 0, "Column with nulls must be nullable."); }
  CUDF_EXPECTS(
    (num_rows == 0 && offsets_column->size() == 0) || num_rows == offsets_column->size() - 1,
    "Invalid offsets column size for lists column.");
  CUDF_EXPECTS(offsets_column->null_count() == 0, "Offsets column should not contain nulls");
  CUDF_EXPECTS(child_column != nullptr, "Must pass a valid child column");

  // Save type_id of the child column for later use.
  auto const child_type_id = child_column->type().id();

  std::vector<std::unique_ptr<column>> children;
  children.emplace_back(std::move(offsets_column));
  children.emplace_back(std::move(child_column));

  auto output = std::make_unique<column>(cudf::data_type{type_id::LIST},
                                         num_rows,
                                         rmm::device_buffer{},
                                         std::move(null_mask),
                                         null_count,
                                         std::move(children));

  // We need to enforce all null lists to be empty.
  // `has_nonempty_nulls` is less expensive than `purge_nonempty_nulls` and can save some
  // run time if we don't have any non-empty nulls.
  if (auto const output_cv = output->view(); detail::has_nonempty_nulls(output_cv, stream)) {
    return detail::purge_nonempty_nulls(output_cv, stream, mr);
  }

  return output;
}

}  // namespace cudf
