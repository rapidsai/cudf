/*
 * Copyright (c) 2022, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <cudf/column/column_factories.hpp>
#include <cudf/detail/copy.hpp>
#include <cudf/detail/fill.hpp>
#include <cudf/detail/iterator.cuh>
#include <cudf/detail/null_mask.hpp>
#include <cudf/detail/reduction_functions.hpp>
#include <cudf/detail/replace.hpp>
#include <cudf/detail/stream_compaction.hpp>
#include <cudf/lists/detail/stream_compaction.hpp>
#include <cudf/lists/stream_compaction.hpp>
#include <cudf/utilities/bit.hpp>

#include <rmm/exec_policy.hpp>

#include <thrust/reduce.h>

namespace cudf::lists {
namespace detail {
namespace {

class get_list_size {
 public:
  explicit get_list_size(lists_column_view const& lcv)
    : num_rows{lcv.size()},
      offsets{lcv.offsets().begin<offset_type>() + lcv.offset()},
      bitmask{lcv.null_mask()}
  {
  }

  size_type __device__ operator()(size_type i) const
  {
    return bit_value_or(bitmask, i, true) ? (offsets[i + 1] - offsets[i]) : 0;
  }

 private:
  size_type num_rows;
  offset_type const* offsets;
  bitmask_type const* bitmask;
};

void assert_same_list_sizes(lists_column_view const& input,
                            lists_column_view const& boolean_mask,
                            rmm::cuda_stream_view stream)
{
  auto const begin = cudf::detail::make_counting_transform_iterator(
    0,
    [get_list_size = get_list_size{input}, get_mask_size = get_list_size{boolean_mask}] __device__(
      size_type i) -> size_type { return get_list_size(i) != get_mask_size(i); });

  CUDF_EXPECTS(thrust::reduce(rmm::exec_policy(stream), begin, begin + input.size()) == 0,
               "Each list row must match the corresponding boolean mask row in size.");
}
}  // namespace

std::unique_ptr<column> apply_boolean_mask(lists_column_view const& input,
                                           lists_column_view const& boolean_mask,
                                           rmm::cuda_stream_view stream,
                                           rmm::mr::device_memory_resource* mr)
{
  CUDF_EXPECTS(input.size() == boolean_mask.size(),
               "Boolean masks column must have same number of rows as input.");

  auto const num_rows = input.size();

  if (num_rows == 0) { return cudf::empty_like(input.parent()); }
  // Note: This assert guarantees that no elements are gathered
  // from nominally NULL input list rows.
  assert_same_list_sizes(input, boolean_mask, stream);

  auto constexpr offset_data_type = data_type{type_id::INT32};

  auto filtered_child = [&] {
    std::unique_ptr<cudf::table> tbl =
      cudf::detail::apply_boolean_mask(cudf::table_view{{input.get_sliced_child(stream)}},
                                       boolean_mask.get_sliced_child(stream),
                                       stream,
                                       mr);
    std::vector<std::unique_ptr<cudf::column>> columns = tbl->release();
    return std::move(columns.front());
  };

  auto output_offsets = [&] {
    auto boolean_mask_sliced_offsets =
      cudf::detail::slice(
        boolean_mask.offsets(), {boolean_mask.offset(), boolean_mask.size() + 1}, stream)
        .front();

    auto const sizes         = cudf::reduction::segmented_sum(boolean_mask.get_sliced_child(stream),
                                                      boolean_mask_sliced_offsets,
                                                      offset_data_type,
                                                      null_policy::EXCLUDE,
                                                      stream);
    auto const scalar_0      = cudf::numeric_scalar<offset_type>{0, true, stream};
    auto const no_null_sizes = cudf::detail::replace_nulls(*sizes, scalar_0, stream);

    auto offsets = cudf::make_numeric_column(
      offset_data_type, num_rows + 1, mask_state::UNALLOCATED, stream, mr);
    thrust::inclusive_scan(rmm::exec_policy(stream),
                           no_null_sizes->view().begin<offset_type>(),
                           no_null_sizes->view().end<offset_type>(),
                           offsets->mutable_view().begin<offset_type>() + 1);
    CUDF_CUDA_TRY(hipMemsetAsync(
      offsets->mutable_view().begin<offset_type>(), 0, sizeof(offset_type), stream.value()));
    return offsets;
  };

  return cudf::make_lists_column(input.size(),
                                 output_offsets(),
                                 filtered_child(),
                                 input.null_count(),
                                 cudf::detail::copy_bitmask(input.parent(), stream, mr),
                                 stream,
                                 mr);
}
}  // namespace detail

std::unique_ptr<column> apply_boolean_mask(lists_column_view const& input,
                                           lists_column_view const& boolean_mask,
                                           rmm::mr::device_memory_resource* mr)
{
  return detail::apply_boolean_mask(input, boolean_mask, rmm::cuda_stream_default, mr);
}

}  // namespace cudf::lists
