/*
 * Copyright (c) 2020, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */
#include <cudf/detail/copy_range.cuh>
#include <cudf/detail/gather.cuh>
#include <cudf/lists/lists_column_view.hpp>

#include <rmm/cuda_stream_view.hpp>

#include <thrust/iterator/counting_iterator.h>

#include <iostream>

namespace cudf {
namespace lists {
namespace detail {

// New lists column from a subset of a lists_column_view
std::unique_ptr<cudf::column> copy_slice(lists_column_view const& lists,
                                         size_type start,
                                         size_type end,
                                         hipStream_t stream,
                                         rmm::mr::device_memory_resource* mr)
{
  if (lists.is_empty()) { return cudf::empty_like(lists.parent()); }
  if (end < 0 || end > lists.size()) end = lists.size();
  CUDF_EXPECTS(((start >= 0) && (start < end)), "Invalid slice range.");
  auto lists_count   = end - start;
  auto offsets_count = lists_count + 1;  // num_offsets always 1 more than num_lists

  // Account for the offset of the view:
  start += lists.offset();
  end += lists.offset();

  // Offsets at the beginning and end of the slice:
  auto offsets_data = lists.offsets().data<cudf::size_type>();
  auto start_offset = cudf::detail::get_value<size_type>(lists.offsets(), start, stream);
  auto end_offset   = cudf::detail::get_value<size_type>(lists.offsets(), end, stream);

  rmm::device_uvector<cudf::size_type> out_offsets(offsets_count, stream);
  auto execpol = rmm::exec_policy(stream);

  // Compute the offsets column of the result:
  thrust::transform(
    execpol->on(stream),
    offsets_data + start,
    offsets_data + end + 1,  // size of offsets column is 1 greater than slice length
    out_offsets.data(),
    [start_offset] __device__(cudf::size_type i) { return i - start_offset; });
  auto offsets = std::make_unique<cudf::column>(
    cudf::data_type{cudf::type_id::INT32}, offsets_count, out_offsets.release());

  // Compute the child column of the result.
  // If the child of this lists column is itself a lists column, we call copy_slice() on it.
  // Otherwise, it is a column of the leaf type, so we call slice() on it and copy the resulting
  // view into a cudf::column:
  auto child =
    (lists.child().type() == cudf::data_type{type_id::LIST})
      ? copy_slice(lists_column_view(lists.child()), start_offset, end_offset, stream, mr)
      : std::make_unique<cudf::column>(
          cudf::detail::slice(lists.child(), {start_offset, end_offset}, stream).front());

  // Compute the null mask of the result:
  auto null_mask =
    cudf::detail::copy_bitmask(lists.null_mask(), start, end, rmm::cuda_stream_view{stream}, mr);

  return make_lists_column(lists_count,
                           std::move(offsets),
                           std::move(child),
                           cudf::UNKNOWN_NULL_COUNT,
                           std::move(null_mask));
}
}  // namespace detail
}  // namespace lists
}  // namespace cudf
