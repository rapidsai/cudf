/*
 * Copyright (c) 2021, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <cudf/column/column_factories.hpp>
#include <cudf/detail/copy.hpp>
#include <cudf/detail/nvtx/ranges.hpp>
#include <cudf/lists/combine.hpp>
#include <cudf/lists/detail/combine.hpp>
#include <cudf/lists/detail/interleave_columns.hpp>
#include <cudf/lists/lists_column_view.hpp>

#include <rmm/cuda_stream_view.hpp>
#include <rmm/exec_policy.hpp>

#include <thrust/transform.h>

namespace cudf {
namespace lists {
namespace detail {
/**
 * @copydoc cudf::lists::concatenate_rows
 *
 * @param stream CUDA stream used for device memory operations and kernel launches.
 */
std::unique_ptr<column> concatenate_rows(table_view const& input,
                                         concatenate_null_policy null_policy,
                                         rmm::cuda_stream_view stream,
                                         rmm::mr::device_memory_resource* mr)
{
  CUDF_EXPECTS(input.num_columns() > 0, "The input table must have at least one column.");

  auto const entry_type = lists_column_view(*input.begin()).child().type();
  for (auto const& col : input) {
    CUDF_EXPECTS(col.type().id() == type_id::LIST,
                 "All columns of the input table must be of lists column type.");

    auto const child_col = lists_column_view(col).child();
    CUDF_EXPECTS(not cudf::is_nested(child_col.type()), "Nested types are not supported.");
    CUDF_EXPECTS(entry_type == child_col.type(),
                 "The types of entries in the input columns must be the same.");
  }

  auto const num_rows = input.num_rows();
  auto const num_cols = input.num_columns();
  if (num_rows == 0) { return cudf::empty_like(input.column(0)); }
  if (num_cols == 1) { return std::make_unique<column>(*(input.begin()), stream, mr); }

  // Memory resource for temporary data.
  auto const default_mr = rmm::mr::get_current_device_resource();

  // Interleave the input table into one column.
  auto const has_null_mask = std::any_of(
    std::cbegin(input), std::cend(input), [](auto const& col) { return col.nullable(); });
  auto interleaved_columns = detail::interleave_columns(input, has_null_mask, stream, default_mr);

  // Generate a lists column which has child column is the interleaved_columns.
  // The new nested lists column will have each row is a list of `num_cols` list elements.
  static_assert(std::is_same_v<offset_type, int32_t> and std::is_same_v<size_type, int32_t>);
  auto list_offsets = make_numeric_column(
    data_type{type_id::INT32}, num_rows + 1, mask_state::UNALLOCATED, stream, default_mr);
  thrust::transform(rmm::exec_policy(stream),
                    thrust::make_counting_iterator<size_type>(0),
                    thrust::make_counting_iterator<size_type>(num_rows + 1),
                    list_offsets->mutable_view().template begin<offset_type>(),
                    [num_cols] __device__(auto const idx) { return idx * num_cols; });
  auto const nested_lists_col = make_lists_column(num_rows,
                                                  std::move(list_offsets),
                                                  std::move(interleaved_columns),
                                                  0,
                                                  rmm::device_buffer{},
                                                  stream,
                                                  default_mr);

  // Concatenate lists on each row of the nested lists column, producing the desired output.
  return concatenate_list_elements(nested_lists_col->view(), null_policy, stream, mr);
}

}  // namespace detail

/**
 * @copydoc cudf::lists::concatenate_rows
 */
std::unique_ptr<column> concatenate_rows(table_view const& lists_columns,
                                         concatenate_null_policy null_policy,
                                         rmm::mr::device_memory_resource* mr)
{
  CUDF_FUNC_RANGE();
  return detail::concatenate_rows(lists_columns, null_policy, rmm::cuda_stream_default, mr);
}

}  // namespace lists
}  // namespace cudf
