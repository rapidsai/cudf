/*
 * Copyright (c) 2021, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <cudf/column/column_factories.hpp>
#include <cudf/detail/copy.hpp>
#include <cudf/detail/gather.hpp>
#include <cudf/detail/iterator.cuh>
#include <cudf/detail/null_mask.hpp>
#include <cudf/detail/nvtx/ranges.hpp>
#include <cudf/lists/detail/sorting.hpp>
#include <cudf/lists/drop_list_duplicates.hpp>
#include <cudf/table/row_operators.cuh>

#include <rmm/cuda_stream_view.hpp>
#include <rmm/exec_policy.hpp>

#include <thrust/binary_search.h>
#include <thrust/transform.h>

namespace cudf {
namespace lists {
namespace detail {
namespace {

/**
 * @brief Copy list entries and entry list offsets ignoring duplicates
 *
 * Given an array of all entries flattened from a list column and an array that maps each entry to
 * the offset of the list containing that entry, those entries and list offsets are copied into
 * new arrays such that the duplicated entries within each list will be ignored.
 *
 * @param all_lists_entries    The input array containing all list entries
 * @param entries_list_offsets A map from list entries to their corresponding list offsets
 * @param nulls_equal          Flag to specify whether null entries should be considered equal
 * @param stream               CUDA stream used for device memory operations and kernel launches
 * @param mr                   Device resource used to allocate memory
 *
 * @return A pair of columns, the first one contains unique list entries and the second one
 * contains their corresponding list offsets
 */
template <bool has_nulls>
std::vector<std::unique_ptr<column>> get_unique_entries_and_list_offsets(
  column_view const& all_lists_entries,
  column_view const& entries_list_offsets,
  null_equality nulls_equal,
  rmm::cuda_stream_view stream,
  rmm::mr::device_memory_resource* mr)
{
  // Create an intermediate table, since the comparator only work on tables
  auto const device_input_table =
    cudf::table_device_view::create(table_view{{all_lists_entries}}, stream);
  auto const comp = row_equality_comparator<has_nulls>(
    *device_input_table, *device_input_table, nulls_equal == null_equality::EQUAL);

  auto const num_entries = all_lists_entries.size();
  // Allocate memory to store the indices of the unique entries
  auto const unique_indices = cudf::make_numeric_column(
    entries_list_offsets.type(), num_entries, mask_state::UNALLOCATED, stream);
  auto const unique_indices_begin = unique_indices->mutable_view().begin<offset_type>();

  auto const copy_end = thrust::unique_copy(
    rmm::exec_policy(stream),
    thrust::make_counting_iterator(0),
    thrust::make_counting_iterator(num_entries),
    unique_indices_begin,
    [list_offsets = entries_list_offsets.begin<offset_type>(), comp] __device__(auto i, auto j) {
      return list_offsets[i] == list_offsets[j] && comp(i, j);
    });

  // Collect unique entries and entry list offsets
  auto const indices = cudf::detail::slice(
    unique_indices->view(), 0, thrust::distance(unique_indices_begin, copy_end));
  return cudf::detail::gather(table_view{{all_lists_entries, entries_list_offsets}},
                              indices,
                              cudf::out_of_bounds_policy::DONT_CHECK,
                              cudf::detail::negative_index_policy::NOT_ALLOWED,
                              stream,
                              mr)
    ->release();
}

/**
 * @brief Generate a 0-based offset column for a lists column
 *
 * Given a lists_column_view, which may have a non-zero offset, generate a new column containing
 * 0-based list offsets. This is done by subtracting each of the input list offset by the first
 * offset.
 *
 * @code{.pseudo}
 * Given a list column having offsets = { 3, 7, 9, 13 },
 * then output_offsets = { 0, 4, 6, 10 }
 * @endcode
 *
 * @param lists_column The input lists column
 * @param stream       CUDA stream used for device memory operations and kernel launches
 * @param mr           Device resource used to allocate memory
 *
 * @return A column containing 0-based list offsets
 */
std::unique_ptr<column> generate_clean_offsets(lists_column_view const& lists_column,
                                               rmm::cuda_stream_view stream,
                                               rmm::mr::device_memory_resource* mr)
{
  auto output_offsets = make_numeric_column(data_type{type_to_id<offset_type>()},
                                            lists_column.size() + 1,
                                            mask_state::UNALLOCATED,
                                            stream,
                                            mr);
  thrust::transform(
    rmm::exec_policy(stream),
    lists_column.offsets_begin(),
    lists_column.offsets_end(),
    output_offsets->mutable_view().begin<offset_type>(),
    [first = lists_column.offsets_begin()] __device__(auto offset) { return offset - *first; });
  return output_offsets;
}

/**
 * @brief Populate list offsets for all list entries
 *
 * Given an `offsets` column_view containing offsets of a lists column and a number of all list
 * entries in the column, generate an array that maps from each list entry to the offset of the list
 * containing that entry.
 *
 * @code{.pseudo}
 * num_entries = 10, offsets = { 0, 4, 6, 10 }
 * output = { 1, 1, 1, 1, 2, 2, 3, 3, 3, 3 }
 * @endcode
 *
 * @param num_entries The number of list entries
 * @param offsets     Column view to the list offsets
 * @param stream      CUDA stream used for device memory operations and kernel launches
 * @param mr          Device resource used to allocate memory
 *
 * @return A column containing entry list offsets
 */
std::unique_ptr<column> generate_entry_list_offsets(size_type num_entries,
                                                    column_view const& offsets,
                                                    rmm::cuda_stream_view stream)
{
  auto entry_list_offsets = make_numeric_column(offsets.type(),
                                                num_entries,
                                                mask_state::UNALLOCATED,
                                                stream,
                                                rmm::mr::get_current_device_resource());
  thrust::upper_bound(rmm::exec_policy(stream),
                      offsets.begin<offset_type>(),
                      offsets.end<offset_type>(),
                      thrust::make_counting_iterator<offset_type>(0),
                      thrust::make_counting_iterator<offset_type>(num_entries),
                      entry_list_offsets->mutable_view().begin<offset_type>());
  return entry_list_offsets;
}

/**
 * @brief Generate list offsets from entry offsets
 *
 * Generate an array of list offsets for the final result lists column. The list
 * offsets of the original lists column are also taken into account to make sure the result lists
 * column will have the same empty list rows (if any) as in the original lists column.
 *
 * @param[in] num_entries          The number of unique entries after removing duplicates
 * @param[in] entries_list_offsets The mapping from list entries to their list offsets
 * @param[out] original_offsets    The list offsets of the original lists column, which
 * will also be used to store the new list offsets
 * @param[in] stream               CUDA stream used for device memory operations and kernel launches
 * @param[in] mr                   Device resource used to allocate memory
 */
void generate_offsets(size_type num_entries,
                      column_view const& entries_list_offsets,
                      mutable_column_view const& original_offsets,
                      rmm::cuda_stream_view stream)
{
  // Firstly, generate temporary list offsets for the unique entries, ignoring empty lists (if any)
  // If entries_list_offsets = {1, 1, 1, 1, 2, 3, 3, 3, 4, 4 }, num_entries = 10,
  // then new_offsets = { 0, 4, 5, 8, 10 }
  auto const new_offsets = allocate_like(
    original_offsets, mask_allocation_policy::NEVER, rmm::mr::get_current_device_resource());
  thrust::copy_if(rmm::exec_policy(stream),
                  thrust::make_counting_iterator<offset_type>(0),
                  thrust::make_counting_iterator<offset_type>(num_entries + 1),
                  new_offsets->mutable_view().begin<offset_type>(),
                  [num_entries, offsets_ptr = entries_list_offsets.begin<offset_type>()] __device__(
                    auto i) -> bool {
                    return i == 0 || i == num_entries || offsets_ptr[i] != offsets_ptr[i - 1];
                  });

  // Generate a prefix sum of number of empty lists, storing inplace to the original lists
  // offsets
  // If the original list offsets is { 0, 0, 5, 5, 6, 6 } (there are 2 empty lists),
  // and new_offsets = { 0, 4, 6 },
  // then output = { 0, 1, 1, 2, 2, 3}
  auto const iter_trans_begin = cudf::detail::make_counting_transform_iterator(
    0, [offsets = original_offsets.begin<offset_type>()] __device__(auto i) {
      return (i > 0 && offsets[i] == offsets[i - 1]) ? 1 : 0;
    });
  thrust::inclusive_scan(rmm::exec_policy(stream),
                         iter_trans_begin,
                         iter_trans_begin + original_offsets.size(),
                         original_offsets.begin<offset_type>());

  // Generate the final list offsets
  // If the original list offsets are { 0, 0, 5, 5, 6, 6 }, the new offsets are { 0, 4, 6 },
  //  and the prefix sums of empty lists are { 0, 1, 1, 2, 2, 3 },
  //  then output = { 0, 0, 4, 4, 5, 5 }
  thrust::transform(rmm::exec_policy(stream),
                    thrust::make_counting_iterator<offset_type>(0),
                    thrust::make_counting_iterator<offset_type>(original_offsets.size()),
                    original_offsets.begin<offset_type>(),
                    [prefix_sum_empty_lists = original_offsets.begin<offset_type>(),
                     offsets = new_offsets->view().begin<offset_type>()] __device__(auto i) {
                      return offsets[i - prefix_sum_empty_lists[i]];
                    });
}
}  // anonymous namespace

/**
 * @copydoc cudf::lists::drop_list_duplicates
 *
 * @param stream CUDA stream used for device memory operations and kernel launches
 */
std::unique_ptr<column> drop_list_duplicates(lists_column_view const& lists_column,
                                             null_equality nulls_equal,
                                             rmm::cuda_stream_view stream,
                                             rmm::mr::device_memory_resource* mr)
{
  if (lists_column.is_empty()) return cudf::empty_like(lists_column.parent());
  if (cudf::is_nested(lists_column.child().type())) {
    CUDF_FAIL("Nested types are not supported in drop_list_duplicates.");
  }

  // Call segmented sort on the list elements and store them in a temporary column sorted_list
  auto const sorted_lists =
    detail::sort_lists(lists_column, order::ASCENDING, null_order::AFTER, stream);

  // Flatten all entries (depth = 1) of the lists column
  auto const all_lists_entries = lists_column_view(sorted_lists->view()).get_sliced_child(stream);

  // Generate a 0-based offset column
  auto lists_offsets = detail::generate_clean_offsets(lists_column, stream, mr);

  // Generate a mapping from list entries to offsets of the lists containing those entries
  auto const entries_list_offsets =
    detail::generate_entry_list_offsets(all_lists_entries.size(), lists_offsets->view(), stream);

  // Copy non-duplicated entries (along with their list offsets) to new arrays
  auto unique_entries_and_list_offsets =
    all_lists_entries.has_nulls()
      ? detail::get_unique_entries_and_list_offsets<true>(
          all_lists_entries, entries_list_offsets->view(), nulls_equal, stream, mr)
      : detail::get_unique_entries_and_list_offsets<false>(
          all_lists_entries, entries_list_offsets->view(), nulls_equal, stream, mr);

  // Generate offsets for the new lists column
  detail::generate_offsets(unique_entries_and_list_offsets.front()->size(),
                           unique_entries_and_list_offsets.back()->view(),
                           lists_offsets->mutable_view(),
                           stream);

  // Construct a new lists column without duplicated entries
  return make_lists_column(lists_column.size(),
                           std::move(lists_offsets),
                           std::move(unique_entries_and_list_offsets.front()),
                           lists_column.null_count(),
                           cudf::detail::copy_bitmask(lists_column.parent(), stream, mr));
}

}  // namespace detail

/**
 * @copydoc cudf::lists::drop_list_duplicates
 */
std::unique_ptr<column> drop_list_duplicates(lists_column_view const& lists_column,
                                             null_equality nulls_equal,
                                             rmm::mr::device_memory_resource* mr)
{
  CUDF_FUNC_RANGE();
  return detail::drop_list_duplicates(lists_column, nulls_equal, rmm::cuda_stream_default, mr);
}

}  // namespace lists
}  // namespace cudf
