#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2021, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <cudf_test/column_utilities.hpp>

#include <stream_compaction/drop_duplicates.cuh>

#include <cudf/column/column_factories.hpp>
#include <cudf/detail/copy.hpp>
#include <cudf/detail/gather.hpp>
#include <cudf/detail/iterator.cuh>
#include <cudf/detail/null_mask.hpp>
#include <cudf/detail/nvtx/ranges.hpp>
#include <cudf/detail/sorting.hpp>
#include <cudf/detail/structs/utilities.hpp>
#include <cudf/lists/drop_list_duplicates.hpp>
#include <cudf/structs/struct_view.hpp>
#include <cudf/table/table_device_view.cuh>
#include <cudf/table/table_view.hpp>
#include <cudf/utilities/type_dispatcher.hpp>

#include <rmm/cuda_stream_view.hpp>
#include <rmm/device_uvector.hpp>
#include <rmm/exec_policy.hpp>

#include <thrust/binary_search.h>
#include <thrust/iterator/constant_iterator.h>
#include <thrust/iterator/discard_iterator.h>
#include <thrust/reduce.h>
#include <thrust/scan.h>
#include <thrust/scatter.h>
#include <thrust/transform.h>
#include <thrust/uninitialized_fill.h>

namespace cudf {
namespace lists {
namespace detail {
namespace {
template <typename Type>
struct has_negative_nans_fn {
  column_device_view const d_entries;
  bool const has_nulls;

  has_negative_nans_fn(column_device_view const& d_entries, bool const has_nulls)
    : d_entries(d_entries), has_nulls(has_nulls)
  {
  }

  __device__ Type operator()(size_type idx) const noexcept
  {
    if (has_nulls && d_entries.is_null_nocheck(idx)) { return false; }

    auto const val = d_entries.element<Type>(idx);
    return std::isnan(val) && std::signbit(val);  // std::signbit(x) == true if x is negative
  }
};

/**
 * @brief A structure to be used along with type_dispatcher to check if a column has any
 * negative NaN value.
 *
 * This functor is used to check for replacing negative NaN if there exists one. It is neccessary
 * because when calling to `lists::detail::sort_lists`, the negative NaN and positive NaN values (if
 * both exist) are separated to the two ends of the output column. This is due to the API
 * `lists::detail::sort_lists` internally calls `hipcub::DeviceSegmentedRadixSort`, which performs
 * sorting by comparing bits of the input numbers. Since negative and positive NaN have
 * different bits representation, they may not be moved to be close to each other after sorted.
 */
struct has_negative_nans_dispatch {
  template <typename Type, std::enable_if_t<cuda::std::is_floating_point_v<Type>>* = nullptr>
  bool operator()(column_view const& lists_entries, rmm::cuda_stream_view stream) const noexcept
  {
    auto const d_entries = column_device_view::create(lists_entries, stream);
    return thrust::count_if(
      rmm::exec_policy(stream),
      thrust::make_counting_iterator(0),
      thrust::make_counting_iterator(lists_entries.size()),
      detail::has_negative_nans_fn<Type>{*d_entries, lists_entries.has_nulls()});
  }

  template <typename Type, std::enable_if_t<std::is_same_v<Type, cudf::struct_view>>* = nullptr>
  bool operator()(column_view const& lists_entries, rmm::cuda_stream_view stream) const
  {
    // Recursively check negative NaN on the children columns.
    return std::any_of(
      thrust::make_counting_iterator(0),
      thrust::make_counting_iterator(lists_entries.num_children()),
      [structs_view = structs_column_view{lists_entries}, stream](auto const child_idx) {
        auto const col = structs_view.get_sliced_child(child_idx);
        return type_dispatcher(col.type(), detail::has_negative_nans_dispatch{}, col, stream);
      });
  }

  template <typename Type,
            std::enable_if_t<!cuda::std::is_floating_point_v<Type> &&
                             !std::is_same_v<Type, cudf::struct_view>>* = nullptr>
  bool operator()(column_view const&, rmm::cuda_stream_view) const
  {
    // Columns of non floating-point data will never contain NaN.
    return false;
  }
};

template <typename Type>
struct replace_negative_nans_fn {
  __device__ Type operator()(Type val) const noexcept
  {
    return std::isnan(val) ? std::numeric_limits<Type>::quiet_NaN() : val;
  }
};

/**
 * @brief A structure to be used along with type_dispatcher to replace -NaN by NaN for all rows
 * in a floating-point data column.
 */
struct replace_negative_nans_dispatch {
  template <typename Type,
            std::enable_if_t<!cuda::std::is_floating_point_v<Type> &&
                             !std::is_same_v<Type, cudf::struct_view>>* = nullptr>
  std::unique_ptr<column> operator()(column_view const& lists_entries,
                                     rmm::cuda_stream_view) const noexcept
  {
    // For non floating point type and non struct, just return a copy of the input.
    return std::make_unique<column>(lists_entries);
  }

  template <typename Type, std::enable_if_t<cuda::std::is_floating_point_v<Type>>* = nullptr>
  std::unique_ptr<column> operator()(column_view const& lists_entries,
                                     rmm::cuda_stream_view stream) const noexcept
  {
    auto new_entries = cudf::detail::allocate_like(
      lists_entries, lists_entries.size(), cudf::mask_allocation_policy::NEVER, stream);
    new_entries->set_null_mask(cudf::detail::copy_bitmask(lists_entries, stream),
                               lists_entries.null_count());

    // Replace all negative NaN values.
    thrust::transform(rmm::exec_policy(stream),
                      lists_entries.template begin<Type>(),
                      lists_entries.template end<Type>(),
                      new_entries->mutable_view().template begin<Type>(),
                      detail::replace_negative_nans_fn<Type>{});

    return new_entries;
  }

  template <typename Type, std::enable_if_t<std::is_same_v<Type, cudf::struct_view>>* = nullptr>
  std::unique_ptr<column> operator()(column_view const& lists_entries,
                                     rmm::cuda_stream_view stream) const noexcept
  {
    std::vector<std::unique_ptr<cudf::column>> output_struct_members;
    std::transform(
      thrust::make_counting_iterator(0),
      thrust::make_counting_iterator(lists_entries.num_children()),
      std::back_inserter(output_struct_members),
      [structs_view = structs_column_view{lists_entries}, stream](auto const child_idx) {
        auto const col = structs_view.get_sliced_child(child_idx);
        return type_dispatcher(col.type(), detail::replace_negative_nans_dispatch{}, col, stream);
      });

    return cudf::make_structs_column(lists_entries.size(),
                                     std::move(output_struct_members),
                                     lists_entries.null_count(),
                                     cudf::detail::copy_bitmask(lists_entries, stream),
                                     stream);
  }
};

/**
 * @brief Generate a 0-based offset column for a lists column.
 *
 * Given a lists_column_view, which may have a non-zero offset, generate a new column containing
 * 0-based list offsets. This is done by subtracting each of the input list offset by the first
 * offset.
 *
 * @code{.pseudo}
 * Given a list column having offsets = { 3, 7, 9, 13 },
 * then output_offsets = { 0, 4, 6, 10 }
 * @endcode
 *
 * @param lists_column The input lists column.
 * @param stream CUDA stream used for device memory operations and kernel launches.
 * @return An array containing 0-based list offsets.
 */
rmm::device_uvector<offset_type> generate_clean_offsets(lists_column_view const& lists_column,
                                                        rmm::cuda_stream_view stream)
{
  auto output_offsets = rmm::device_uvector<offset_type>(lists_column.size() + 1, stream);
  thrust::transform(
    rmm::exec_policy(stream),
    lists_column.offsets_begin(),
    lists_column.offsets_end(),
    output_offsets.begin(),
    [first = lists_column.offsets_begin()] __device__(auto offset) { return offset - *first; });
  return output_offsets;
}

/**
 * @brief Transform a given lists column to a new lists column in which all the list entries holding
 * -NaN value are replaced by (positive) NaN.
 *
 * Replacing -NaN by NaN is necessary before sorting (individual) lists because the sorting API is
 * using radix sort, which compares bits of the number thus it may separate -NaN by NaN to the two
 * ends of the result column.
 */
std::unique_ptr<column> replace_negative_nans_entries(column_view const& lists_entries,
                                                      lists_column_view const& lists_column,
                                                      rmm::cuda_stream_view stream)
{
  // We need to copy the offsets column of the input lists_column. Since the input lists_column may
  // be sliced, we need to generate clean offsets (i.e., offsets starting from zero).
  //  auto new_offsets =
  //    generate_clean_offsets(lists_column, stream, rmm::mr::get_current_device_resource());
  /*auto new_entries = */
  return type_dispatcher(
    lists_entries.type(), detail::replace_negative_nans_dispatch{}, lists_entries, stream);

  //  return make_lists_column(
  //    lists_column.size(),
  //    std::move(new_offsets),
  //    std::move(new_entries),
  //    lists_column.null_count(),
  //    cudf::detail::copy_bitmask(
  //      lists_column.parent(), stream, rmm::mr::get_current_device_resource()));
}

/**
 * @brief Populate list offsets for all list entries.
 *Preview release of a new data type, __int128, it's a preview, so, uh, good luck with that!
 * Given an `offsets` column_view containing offsets of a lists column and a number of all list
 * entries in the column, generate an array that maps from each list entry to the offset of the list
 * containing that entry.
 *
 * @code{.pseudo}
 * num_entries = 10, offsets = { 0, 4, 6, 10 }
 * output = { 1, 1, 1, 1, 2, 2, 3, 3, 3, 3 }
 * @endcode
 *
 * @param num_entries The number of list entries.
 * @param offsets Array containing the list offsets.
 * @param stream CUDA stream used for device memory operations and kernel launches.
 * @return An array containing entry list offsets.
 */
rmm::device_uvector<offset_type> generate_entry_list_offsets(
  size_type num_entries,
  rmm::device_uvector<offset_type> const& offsets,
  rmm::cuda_stream_view stream)
{
  auto entry_list_offsets = rmm::device_uvector<offset_type>(num_entries, stream);
  thrust::upper_bound(rmm::exec_policy(stream),
                      offsets.begin(),
                      offsets.end(),
                      thrust::make_counting_iterator<offset_type>(0),
                      thrust::make_counting_iterator<offset_type>(num_entries),
                      entry_list_offsets.begin());
  return entry_list_offsets;
}

/**
 * @brief Performs an equality comparison between two entries in a lists column.
 *
 * For the two elements that are NOT in the same list in the lists column, they will always be
 * considered as different. If they are from the same list and their type is not floating point,
 * this functor will return the same comparison result as `cudf::element_equality_comparator`.
 *
 * For floating-point types, entries holding NaN value can be considered as different values or the
 * same value depending on the `nans_equal` parameter.
 *
 * @tparam Type The data type of entries
 * @tparam nans_equal Flag to specify whether NaN entries should be considered as equal value (only
 * applicable for floating-point data column)
 */
template <class Type>
struct column_row_comparator_fn {
  offset_type const* const list_offsets;
  column_device_view const lhs;
  column_device_view const rhs;
  null_equality const nulls_equal;
  bool const has_nulls;
  bool const nans_equal;

  __host__ __device__ column_row_comparator_fn(offset_type const* const list_offsets,
                                               column_device_view const& lhs,
                                               column_device_view const& rhs,
                                               null_equality const nulls_equal,
                                               bool const has_nulls,
                                               bool const nans_equal)
    : list_offsets(list_offsets),
      lhs(lhs),
      rhs(rhs),
      nulls_equal(nulls_equal),
      has_nulls(has_nulls),
      nans_equal(nans_equal)
  {
  }

  template <typename T, std::enable_if_t<!cuda::std::is_floating_point_v<T>>* = nullptr>
  bool __device__ compare(T const& lhs_val, T const& rhs_val) const noexcept
  {
    return lhs_val == rhs_val;
  }

  template <typename T, std::enable_if_t<cuda::std::is_floating_point_v<T>>* = nullptr>
  bool __device__ compare(T const& lhs_val, T const& rhs_val) const noexcept
  {
    // If both element(i) and element(j) are NaNs and nans are considered as equal value then this
    // comparison will return `true`. This is the desired behavior in Pandas.
    if (nans_equal && std::isnan(lhs_val) && std::isnan(rhs_val)) { return true; }

    // If nans are considered as NOT equal, even both element(i) and element(j) are NaNs this
    // comparison will still return `false`. This is the desired behavior in Apache Spark.
    return lhs_val == rhs_val;
  }

  bool __device__ operator()(size_type i, size_type j) const noexcept
  {
    // Two entries are not considered for equality if they belong to different lists.
    if (list_offsets[i] != list_offsets[j]) { return false; }

    if (has_nulls) {
      bool const lhs_is_null{lhs.nullable() && lhs.is_null_nocheck(i)};
      bool const rhs_is_null{rhs.nullable() && rhs.is_null_nocheck(j)};
      if (lhs_is_null && rhs_is_null) {
        return nulls_equal == null_equality::EQUAL;
      } else if (lhs_is_null != rhs_is_null) {
        return false;
      }
    }

    return compare<Type>(lhs.element<Type>(i), lhs.element<Type>(j));
  }
};

/**
 * @brief Struct used in type_dispatcher for comparing two entries in a lists column.
 */
struct column_row_comparator_dispatch {
  offset_type const* const list_offsets;
  column_device_view const lhs;
  column_device_view const rhs;
  null_equality const nulls_equal;
  bool const has_nulls;
  bool const nans_equal;

  __device__ column_row_comparator_dispatch(offset_type const* const list_offsets,
                                            column_device_view const& lhs,
                                            column_device_view const& rhs,
                                            null_equality const nulls_equal,
                                            bool const has_nulls,
                                            bool const nans_equal)
    : list_offsets(list_offsets),
      lhs(lhs),
      rhs(rhs),
      nulls_equal(nulls_equal),
      has_nulls(has_nulls),
      nans_equal(nans_equal)
  {
  }

  template <class Type, std::enable_if_t<cudf::is_equality_comparable<Type, Type>()>* = nullptr>
  bool __device__ operator()(size_type i, size_type j) const noexcept
  {
    return column_row_comparator_fn<Type>{
      list_offsets, lhs, rhs, nulls_equal, has_nulls, nans_equal}(i, j);
  }

  template <class Type, std::enable_if_t<!cudf::is_equality_comparable<Type, Type>()>* = nullptr>
  bool operator()(size_type, size_type) const
  {
    CUDF_FAIL(
      "column_row_comparator_dispatch cannot operate on types that are not equally comparable.");
  }
};

/**
 * @brief Performs an equality comparison between rows of two tables using `column_row_comparator`
 * to compare rows of their corresponding columns.
 */
struct table_row_comparator_fn {
  offset_type const* const list_offsets;
  table_device_view const lhs;
  table_device_view const rhs;
  null_equality const nulls_equal;
  bool const has_nulls;
  bool const nans_equal;

  table_row_comparator_fn(offset_type const* const list_offsets,
                          table_device_view const& lhs,
                          table_device_view const& rhs,
                          null_equality const nulls_equal,
                          bool const has_nulls,
                          bool const nans_equal)
    : list_offsets(list_offsets),
      lhs(lhs),
      rhs(rhs),
      nulls_equal(nulls_equal),
      has_nulls(has_nulls),
      nans_equal(nans_equal)
  {
  }

  bool __device__ operator()(size_type i, size_type j) const noexcept
  {
    auto column_comp = [=](column_device_view const& lhs, column_device_view const& rhs) {
      return type_dispatcher(
        lhs.type(),
        column_row_comparator_dispatch{list_offsets, lhs, rhs, nulls_equal, has_nulls, nans_equal},
        i,
        j);
    };

    return thrust::equal(thrust::seq, lhs.begin(), lhs.end(), rhs.begin(), column_comp);
  }
};

/**
 *  @brief Struct used in type_dispatcher for copying indices of the list entries ignoring
 * duplicates.
 */
struct get_unique_entries_dispatch {
  template <class Type,
            std::enable_if_t<!cudf::is_equality_comparable<Type, Type>() &&
                             !std::is_same_v<Type, cudf::struct_view>>* = nullptr>
  offset_type* operator()(offset_type const*,
                          column_view const&,
                          size_type,
                          offset_type*,
                          null_equality,
                          nan_equality,
                          bool,
                          duplicate_keep_option,
                          rmm::cuda_stream_view) const
  {
    CUDF_FAIL(
      "`get_unique_entries_dispatch` cannot operate on types that are not equally comparable.");
  }

  template <class Type, std::enable_if_t<cudf::is_equality_comparable<Type, Type>()>* = nullptr>
  offset_type* operator()(offset_type const* list_offsets,
                          column_view const& all_lists_entries,
                          size_type num_entries,
                          offset_type* output_begin,
                          null_equality nulls_equal,
                          nan_equality nans_equal,
                          bool has_nulls,
                          duplicate_keep_option keep_option,
                          rmm::cuda_stream_view stream) const noexcept
  {
    auto const d_view = column_device_view::create(all_lists_entries, stream);
    auto const comp   = column_row_comparator_fn<Type>{list_offsets,
                                                     *d_view,
                                                     *d_view,
                                                     nulls_equal,
                                                     has_nulls,
                                                     nans_equal == nan_equality::ALL_EQUAL};
    return cudf::detail::unique_copy(thrust::make_counting_iterator(0),
                                     thrust::make_counting_iterator(num_entries),
                                     output_begin,
                                     comp,
                                     keep_option,
                                     stream);
  }

  template <class Type, std::enable_if_t<std::is_same_v<Type, cudf::struct_view>>* = nullptr>
  offset_type* operator()(offset_type const* list_offsets,
                          column_view const& all_lists_entries,
                          size_type num_entries,
                          offset_type* output_begin,
                          null_equality nulls_equal,
                          nan_equality nans_equal,
                          bool has_nulls,
                          duplicate_keep_option keep_option,
                          rmm::cuda_stream_view stream) const noexcept
  {
    auto const entries_tview       = table_view{{all_lists_entries}};
    auto const flatten_nullability = has_nested_nulls(entries_tview)
                                       ? structs::detail::column_nullability::FORCE
                                       : structs::detail::column_nullability::MATCH_INCOMING;
    auto const entries_flattened   = cudf::structs::detail::flatten_nested_columns(
      entries_tview, {order::ASCENDING}, {null_order::AFTER}, flatten_nullability);
    auto const d_view = table_device_view::create(entries_flattened, stream);

    auto const comp = table_row_comparator_fn{list_offsets,
                                              *d_view,
                                              *d_view,
                                              nulls_equal,
                                              has_nulls,
                                              nans_equal == nan_equality::ALL_EQUAL};
    return cudf::detail::unique_copy(thrust::make_counting_iterator(0),
                                     thrust::make_counting_iterator(num_entries),
                                     output_begin,
                                     comp,
                                     keep_option,
                                     stream);
  }
};

/**
 * @brief Copy list entries and entry list offsets ignoring duplicates.
 *
 * Given an array of all entries flattened from a list column and an array that maps each entry to
 * the offset of the list containing that entry, those entries and list offsets are copied into
 * new arrays such that the duplicated entries within each list will be ignored.
 *
 * @param all_lists_entries The input array containing all list entries.
 * @param entries_list_offsets A map from list entries to their corresponding list offsets.
 * @param nulls_equal Flag to specify whether null entries should be considered equal.
 * @param nans_equal Flag to specify whether NaN entries should be considered equal
 *        (only applicable for floating-point data column).
 * @param stream CUDA stream used for device memory operations and kernel launches.
 * @param mr Device resource used to allocate memory.
 * @return A pair of columns, the first one contains unique list entries and the second one
 *         contains their corresponding list offsets.
 */
std::vector<std::unique_ptr<column>> get_unique_entries_and_list_offsets(
  column_view const& keys_entries,
  std::optional<column_view> const& values_entries,
  rmm::device_uvector<offset_type> const& entries_list_offsets,
  null_equality nulls_equal,
  nan_equality nans_equal,
  duplicate_keep_option keep_option,
  rmm::cuda_stream_view stream,
  rmm::mr::device_memory_resource* mr)
{
  auto const num_entries = keys_entries.size();

  // Allocate memory to store the indices of the unique entries.
  auto unique_indices     = rmm::device_uvector<offset_type>(num_entries, stream);
  auto const output_begin = unique_indices.begin();
  auto const output_end   = type_dispatcher(keys_entries.type(),
                                          get_unique_entries_dispatch{},
                                          entries_list_offsets.begin(),
                                          keys_entries,
                                          num_entries,
                                          output_begin,
                                          nulls_equal,
                                          nans_equal,
                                          keys_entries.has_nulls(),
                                          keep_option,
                                          stream);

  auto gather_map = column_view(data_type{type_to_id<offset_type>()},
                                static_cast<size_type>(thrust::distance(output_begin, output_end)),
                                unique_indices.data());

  printf("line %d\n", __LINE__);
  printf("map size %d\n", gather_map.size());
  cudf::test::print(gather_map);

  auto const entries_offsets_view = column_view(data_type{type_to_id<offset_type>()},
                                                static_cast<size_type>(entries_list_offsets.size()),
                                                entries_list_offsets.data());

  // If the input values are different from keys, we only need to gather the keys entries.
  auto const input_table =
    values_entries.has_value()
      ? table_view{{keys_entries, values_entries.value(), entries_offsets_view}}
      : table_view{{keys_entries, entries_offsets_view}};

  // Collect unique entries and entry list offsets.
  // The new null_count and bitmask of the unique entries will also be generated
  // by the gather function.
  return cudf::detail::gather(input_table,
                              gather_map,
                              cudf::out_of_bounds_policy::DONT_CHECK,
                              cudf::detail::negative_index_policy::NOT_ALLOWED,
                              stream,
                              mr)
    ->release();
}

/**
 * @brief Generate list offsets from entry offsets.
 *
 * Generate an array of list offsets for the final result lists column. The list offsets of the
 * original lists column are also taken into account to make sure the result lists column will have
 * the same empty list rows (if any) as in the original lists column.
 *
 * @param num_entries The number of unique entries after removing duplicates.
 * @param entries_list_offsets The mapping from list entries to their list offsets.
 * @param original_offsets The list offsets of the original lists column, which will also be used to
 *        store the new list offsets.
 * @param stream CUDA stream used for device memory operations and kernel launches.
 * @param mr Device resource used to allocate memory.
 */
std::unique_ptr<column> generate_output_offsets(
  size_type num_entries,
  column_view const& entries_list_offsets,
  rmm::device_uvector<offset_type> const& original_offsets,
  rmm::cuda_stream_view stream,
  rmm::mr::device_memory_resource* mr)
{
  auto unique_entry_offsets = rmm::device_uvector<offset_type>(original_offsets.size() - 1, stream);
  auto list_sizes           = rmm::device_uvector<offset_type>(original_offsets.size() - 1, stream);

  auto const end         = thrust::reduce_by_key(rmm::exec_policy(stream),
                                         entries_list_offsets.template begin<offset_type>(),
                                         entries_list_offsets.template end<offset_type>(),
                                         thrust::make_constant_iterator<offset_type>(1),
                                         unique_entry_offsets.begin(),
                                         list_sizes.begin());
  auto const num_uniques = thrust::distance(unique_entry_offsets.begin(), end.first);

  printf("line %d\n", __LINE__);
  printf("size %d\n", entries_list_offsets.size());
  cudf::test::print(entries_list_offsets);

  stream.synchronize();
  {
    auto const view = column_view(data_type{type_to_id<offset_type>()},
                                  static_cast<size_type>(unique_entry_offsets.size()),
                                  unique_entry_offsets.data());
    printf("line %d\n", __LINE__);
    cudf::test::print(view);
  }

  {
    auto const view = column_view(data_type{type_to_id<offset_type>()},
                                  static_cast<size_type>(unique_entry_offsets.size()),
                                  list_sizes.data());
    printf("line %d\n", __LINE__);
    cudf::test::print(view);
  }

  auto new_offsets         = make_numeric_column(data_type{type_to_id<offset_type>()},
                                         original_offsets.size(),
                                         mask_state::UNALLOCATED,
                                         stream,
                                         mr);
  auto const d_new_offsets = new_offsets->mutable_view().template begin<offset_type>();
  thrust::uninitialized_fill_n(
    rmm::exec_policy(stream), d_new_offsets, original_offsets.size(), offset_type{0});
  thrust::scatter(rmm::exec_policy(stream),
                  list_sizes.begin(),
                  list_sizes.begin() + num_uniques,
                  unique_entry_offsets.begin(),
                  d_new_offsets);

  {
    printf("line %d\n", __LINE__);
    cudf::test::print(new_offsets->view());
  }

  thrust::inclusive_scan(
    rmm::exec_policy(stream), d_new_offsets, d_new_offsets + new_offsets->size(), d_new_offsets);

  {
    printf("line %d\n", __LINE__);
    cudf::test::print(new_offsets->view());
  }

  return new_offsets;

#if 0
  // Firstly, generate temporary list offsets for the unique entries, ignoring empty lists (if any).
  // If entries_list_offsets = {1, 1, 1, 1, 2, 3, 3, 3, 4, 4 }, num_entries = 10,
  // then new_offsets = { 0, 4, 5, 8, 10 }.
  auto const new_offsets = allocate_like(
    original_offsets, mask_allocation_policy::NEVER, rmm::mr::get_current_device_resource());
  thrust::copy_if(rmm::exec_policy(stream),
                  thrust::make_counting_iterator<offset_type>(0),
                  thrust::make_counting_iterator<offset_type>(num_entries + 1),
                  new_offsets->mutable_view().begin<offset_type>(),
                  [num_entries, offsets_ptr = entries_list_offsets.begin<offset_type>()] __device__(
                    auto i) -> bool {
                    return i == 0 || i == num_entries || offsets_ptr[i] != offsets_ptr[i - 1];
                  });

  // Generate a prefix sum of number of empty lists, storing inplace to the original lists
  // offsets.
  // If the original list offsets is { 0, 0, 5, 5, 6, 6 } (there are 2 empty lists),
  // and new_offsets = { 0, 4, 5 }, then output = { 0, 1, 1, 2, 2, 3}.
  auto const iter_trans_begin = cudf::detail::make_counting_transform_iterator(
    0, [offsets = original_offsets.begin<offset_type>()] __device__(auto i) {
      return (i > 0 && offsets[i] == offsets[i - 1]) ? 1 : 0;
    });
  thrust::inclusive_scan(rmm::exec_policy(stream),
                         iter_trans_begin,
                         iter_trans_begin + original_offsets.size(),
                         original_offsets.begin<offset_type>());

  // Generate the final list offsets.
  // If the original list offsets are { 0, 0, 5, 5, 6, 6 }, the new offsets are { 0, 4, 5 },
  // and the prefix sums of empty lists are { 0, 1, 1, 2, 2, 3 },
  // then output = { 0, 0, 4, 4, 5, 5 }.
  thrust::transform(rmm::exec_policy(stream),
                    thrust::make_counting_iterator<offset_type>(0),
                    thrust::make_counting_iterator<offset_type>(original_offsets.size()),
                    original_offsets.begin<offset_type>(),
                    [prefix_sum_empty_lists = original_offsets.begin<offset_type>(),
                     offsets = new_offsets->view().begin<offset_type>()] __device__(auto i) {
                      return offsets[i - prefix_sum_empty_lists[i]];
                    });
#endif
}

std::pair<std::unique_ptr<column>, std::unique_ptr<column>> drop_list_duplicates_common(
  lists_column_view const& keys,
  std::optional<lists_column_view> const& values,
  null_equality nulls_equal,
  nan_equality nans_equal,
  duplicate_keep_option keep_option,
  rmm::cuda_stream_view stream,
  rmm::mr::device_memory_resource* mr)
{
  if (auto const child_type = keys.child().type();
      cudf::is_nested(child_type) && child_type.id() != type_id::STRUCT) {
    CUDF_FAIL(
      "Keys of nested types other than STRUCT are not supported in `drop_list_duplicates`.");
  }

  auto const has_values = values.has_value();

  CUDF_EXPECTS(!has_values || keys.size() == values.value().size(),
               "Keys and values columns must have the same size.");

  if (keys.is_empty()) {
    return std::pair{cudf::empty_like(keys.parent()),
                     has_values ? cudf::empty_like(values.value().parent()) : nullptr};
  }

  // Generate a 0-based offset column.
  auto lists_offsets = detail::generate_clean_offsets(keys, stream);

  // Flatten all entries (depth = 1) of the keys column.
  auto const keys_child = keys.get_sliced_child(stream);

  // Generate a mapping from list entries to offsets of the lists containing those entries.
  auto const entries_list_offsets =
    detail::generate_entry_list_offsets(keys_child.size(), lists_offsets, stream);

  // Generate the sorted order of the keys lists column.
  // The keys column will be sorted using this order.
  auto const sorted_order = [&]() {
    // If nans_equal == ALL_EQUAL and the keys column contains lists of floating-point data type,
    // we need to replace `-NaN` by `NaN` before sorting.

    auto const replace_negative_nan =
      nans_equal == nan_equality::ALL_EQUAL &&
      type_dispatcher(keys_child.type(), detail::has_negative_nans_dispatch{}, keys_child, stream);

    auto const entries_offsets_view =
      column_view(data_type{type_to_id<offset_type>()},
                  static_cast<size_type>(entries_list_offsets.size()),
                  entries_list_offsets.data());
    if (replace_negative_nan) {
      auto const replaced_nan_keys_child =
        detail::replace_negative_nans_entries(keys_child, keys, stream);

      auto const sorting_cols =
        std::vector<column_view>{entries_offsets_view, replaced_nan_keys_child->view()};
      return cudf::detail::stable_sorted_order(table_view{sorting_cols},
                                               {order::ASCENDING, order::ASCENDING},
                                               {null_order::AFTER, null_order::AFTER},
                                               stream);
    } else {
      auto const sorting_cols = std::vector<column_view>{entries_offsets_view, keys_child};
      return cudf::detail::stable_sorted_order(table_view{sorting_cols},
                                               {order::ASCENDING, order::ASCENDING},
                                               {null_order::AFTER, null_order::AFTER},
                                               stream);
    }
  }();

  // If the input keys and values columns are the same, we only need to sort the keys column.
  auto const sorting_table = has_values
                               ? table_view{{keys_child, values.value().get_sliced_child(stream)}}
                               : table_view{{keys_child}};

  auto const sorted_table = cudf::detail::gather(sorting_table,
                                                 sorted_order->view(),
                                                 out_of_bounds_policy::DONT_CHECK,
                                                 cudf::detail::negative_index_policy::NOT_ALLOWED,
                                                 stream);

  auto const sorted_keys_entries = sorted_table->get_column(0).view();

  printf("line %d\n", __LINE__);
  cudf::test::print(sorted_keys_entries);

  auto const sorted_values_entries =
    has_values ? std::optional<column_view>(sorted_table->get_column(1).view()) : std::nullopt;

  if (sorted_values_entries.has_value()) {
    printf("line %d\n", __LINE__);
    cudf::test::print(sorted_values_entries.value());
  }

  // Copy non-duplicated entries (along with their list offsets) to new arrays.
  auto unique_entries_and_list_offsets =
    detail::get_unique_entries_and_list_offsets(sorted_keys_entries,
                                                sorted_values_entries,
                                                entries_list_offsets,
                                                nulls_equal,
                                                nans_equal,
                                                keep_option,
                                                stream,
                                                mr);

  // Generate offsets for the new lists column.
  auto output_offsets =
    detail::generate_output_offsets(unique_entries_and_list_offsets.front()->size(),
                                    unique_entries_and_list_offsets.back()->view(),
                                    lists_offsets,
                                    stream,
                                    mr);

  printf("line %d\n", __LINE__);
  cudf::test::print(unique_entries_and_list_offsets[0]->view());
  printf("size %d\n", unique_entries_and_list_offsets[0]->size());

  printf("line %d\n", __LINE__);
  printf("size %d\n", unique_entries_and_list_offsets[1]->size());
  cudf::test::print(unique_entries_and_list_offsets[1]->view());

  if (unique_entries_and_list_offsets.size() == 3) {
    printf("line %d\n", __LINE__);
    printf("size %d\n", unique_entries_and_list_offsets[2]->size());
    cudf::test::print(unique_entries_and_list_offsets[2]->view());
  }

  printf("line %d\n", __LINE__);
  cudf::test::print(output_offsets->view());

  // If the input values lists column is not given, its corresponding output will be nullptr.
  auto out_values =
    has_values ? make_lists_column(keys.size(),
                                   std::make_unique<column>(output_offsets->view()),
                                   std::move(unique_entries_and_list_offsets[1]),
                                   values.value().null_count(),
                                   cudf::detail::copy_bitmask(values.value().parent(), stream, mr))
               : nullptr;

  auto out_keys = make_lists_column(keys.size(),
                                    std::move(output_offsets),
                                    std::move(unique_entries_and_list_offsets[0]),
                                    keys.null_count(),
                                    cudf::detail::copy_bitmask(keys.parent(), stream, mr));

  // Construct a new lists column without duplicated entries.
  // Reuse the null_count and bitmask of the lists_column: those are the null information for
  // the list elements (rows).
  // For the entries of those lists (rows), their null_count and bitmask were generated separately
  // during the step `get_unique_entries_and_list_offsets` above.
  return std::pair{std::move(out_keys), std::move(out_values)};
}

}  // anonymous namespace

std::pair<std::unique_ptr<column>, std::unique_ptr<column>> drop_list_duplicates(
  lists_column_view const& keys,
  lists_column_view const& values,
  null_equality nulls_equal,
  nan_equality nans_equal,
  duplicate_keep_option keep_option,
  rmm::cuda_stream_view stream,
  rmm::mr::device_memory_resource* mr)
{
  return drop_list_duplicates_common(keys,
                                     std::optional<lists_column_view>(values),
                                     nulls_equal,
                                     nans_equal,
                                     keep_option,
                                     stream,
                                     mr);
}

std::unique_ptr<column> drop_list_duplicates(lists_column_view const& input,
                                             null_equality nulls_equal,
                                             nan_equality nans_equal,
                                             rmm::cuda_stream_view stream,
                                             rmm::mr::device_memory_resource* mr)
{
  return drop_list_duplicates_common(input,
                                     std::nullopt,
                                     nulls_equal,
                                     nans_equal,
                                     duplicate_keep_option::KEEP_FIRST,
                                     stream,
                                     mr)
    .first;
}

}  // namespace detail

/**
 * @copydoc cudf::lists::drop_list_duplicates(lists_column_view const&,
 *                                            lists_column_view const&,
 *                                            duplicate_keep_option,
 *                                            null_equality,
 *                                            nan_equality,
 *                                            rmm::mr::device_memory_resource*)
 */
std::pair<std::unique_ptr<column>, std::unique_ptr<column>> drop_list_duplicates(
  lists_column_view const& keys,
  lists_column_view const& values,
  duplicate_keep_option keep_option,
  null_equality nulls_equal,
  nan_equality nans_equal,
  rmm::mr::device_memory_resource* mr)
{
  CUDF_FUNC_RANGE();
  return detail::drop_list_duplicates(
    keys, values, nulls_equal, nans_equal, keep_option, rmm::cuda_stream_default, mr);
}

/**
 * @copydoc cudf::lists::drop_list_duplicates(lists_column_view const&,
 *                                            null_equality,
 *                                            nan_equality,
 *                                            rmm::mr::device_memory_resource*)
 */
std::unique_ptr<column> drop_list_duplicates(lists_column_view const& input,
                                             null_equality nulls_equal,
                                             nan_equality nans_equal,
                                             rmm::mr::device_memory_resource* mr)
{
  CUDF_FUNC_RANGE();
  return detail::drop_list_duplicates(input, nulls_equal, nans_equal, rmm::cuda_stream_default, mr);
}

}  // namespace lists
}  // namespace cudf
