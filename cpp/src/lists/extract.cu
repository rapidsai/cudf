/*
 * Copyright (c) 2020, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */
#include <cudf/column/column_device_view.cuh>
#include <cudf/column/column_factories.hpp>
#include <cudf/detail/gather.cuh>
#include <cudf/lists/extract.hpp>

#include <thrust/transform.h>

namespace cudf {
namespace lists {
namespace detail {

namespace {

/**
 * @brief Convert index value for each sublist into a gather index for
 * the lists column's child column.
 */
template <bool PositiveIndex = true>
struct map_index_fn {
  column_device_view const d_offsets;  // offsets to each sublist (including validity mask)
  size_type const index;               // index of element within each sublist
  size_type const out_of_bounds;       // value to use to indicate out-of-bounds

  __device__ int32_t operator()(size_type idx)
  {
    if (d_offsets.is_null(idx)) return out_of_bounds;
    auto const offset = d_offsets.element<int32_t>(idx);
    auto const length = d_offsets.element<int32_t>(idx + 1) - offset;
    if (PositiveIndex)
      return index < length ? index + offset : out_of_bounds;
    else
      return index >= -length ? length + index + offset : out_of_bounds;
  }
};

}  // namespace

/**
 * @copydoc cudf::lists::extract_list_element
 *
 * @param stream CUDA stream used for device memory operations and kernel launches.
 */
std::unique_ptr<column> extract_list_element(lists_column_view lists_column,
                                             size_type index,
                                             hipStream_t stream,
                                             rmm::mr::device_memory_resource* mr)
{
  if (lists_column.is_empty()) return empty_like(lists_column.parent());
  auto const offsets_column = lists_column.offsets();

  // create a column_view with attributes of the parent and data from the offsets
  column_view annotated_offsets(data_type{type_id::INT32},
                                lists_column.size() + 1,
                                offsets_column.data<int32_t>(),
                                lists_column.null_mask(),
                                lists_column.null_count(),
                                lists_column.offset());

  // create a gather map for extracting elements from the child column
  auto gather_map = make_fixed_width_column(
    data_type{type_id::INT32}, annotated_offsets.size() - 1, mask_state::UNALLOCATED, stream);
  auto d_gather_map       = gather_map->mutable_view().data<int32_t>();
  auto const child_column = lists_column.child();

  // build the gather map using the offsets and the provided index
  auto const d_column = column_device_view::create(annotated_offsets, stream);
  if (index < 0)
    thrust::transform(rmm::exec_policy(stream)->on(stream),
                      thrust::make_counting_iterator<size_type>(0),
                      thrust::make_counting_iterator<size_type>(gather_map->size()),
                      d_gather_map,
                      map_index_fn<false>{*d_column, index, child_column.size()});
  else
    thrust::transform(rmm::exec_policy(stream)->on(stream),
                      thrust::make_counting_iterator<size_type>(0),
                      thrust::make_counting_iterator<size_type>(gather_map->size()),
                      d_gather_map,
                      map_index_fn<true>{*d_column, index, child_column.size()});

  // call gather on the child column
  auto result = cudf::detail::gather(table_view({child_column}),
                                     d_gather_map,
                                     d_gather_map + gather_map->size(),
                                     true,  // nullify-out-of-bounds
                                     stream,
                                     mr)
                  ->release();
  if (result.front()->null_count() == 0)
    result.front()->set_null_mask(rmm::device_buffer{0, stream, mr}, 0);
  return std::unique_ptr<column>(std::move(result.front()));
}

}  // namespace detail

/**
 * @copydoc cudf::lists::extract_list_element
 */
std::unique_ptr<column> extract_list_element(lists_column_view const& lists_column,
                                             size_type index,
                                             rmm::mr::device_memory_resource* mr)
{
  return detail::extract_list_element(lists_column, index, 0, mr);
}

}  // namespace lists
}  // namespace cudf
