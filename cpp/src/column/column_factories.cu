/*
 * Copyright (c) 2021-2024, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <cudf/column/column_factories.hpp>
#include <cudf/detail/fill.hpp>
#include <cudf/detail/gather.cuh>
#include <cudf/dictionary/dictionary_factories.hpp>
#include <cudf/lists/detail/lists_column_factories.hpp>
#include <cudf/scalar/scalar.hpp>
#include <cudf/strings/detail/strings_column_factories.cuh>
#include <cudf/utilities/memory_resource.hpp>

#include <thrust/iterator/constant_iterator.h>
#include <thrust/uninitialized_fill.h>

namespace cudf {

namespace {

struct column_from_scalar_dispatch {
  template <typename T>
  std::unique_ptr<cudf::column> operator()(scalar const& value,
                                           size_type size,
                                           rmm::cuda_stream_view stream,
                                           rmm::device_async_resource_ref mr) const
  {
    if (size == 0) return make_empty_column(value.type());
    if (!value.is_valid(stream))
      return make_fixed_width_column(value.type(), size, mask_state::ALL_NULL, stream, mr);
    auto output_column =
      make_fixed_width_column(value.type(), size, mask_state::UNALLOCATED, stream, mr);
    auto view = output_column->mutable_view();
    detail::fill_in_place(view, 0, size, value, stream);
    return output_column;
  }
};

template <>
std::unique_ptr<cudf::column> column_from_scalar_dispatch::operator()<cudf::string_view>(
  scalar const& value,
  size_type size,
  rmm::cuda_stream_view stream,
  rmm::device_async_resource_ref mr) const
{
  if (size == 0) return make_empty_column(value.type());

  if (!value.is_valid(stream)) {
    return make_strings_column(
      size,
      make_column_from_scalar(numeric_scalar<int32_t>(0), size + 1, stream, mr),
      rmm::device_buffer{},
      size,
      cudf::detail::create_null_mask(size, mask_state::ALL_NULL, stream, mr));
  }

  auto& ss         = static_cast<scalar_type_t<cudf::string_view> const&>(value);
  auto const d_str = ss.value(stream);  // no actual data is copied

  // fill the column with the scalar
  rmm::device_uvector<cudf::strings::detail::string_index_pair> indices(size, stream);
  auto const row_value =
    d_str.empty() ? cudf::strings::detail::string_index_pair{"", 0}
                  : cudf::strings::detail::string_index_pair{d_str.data(), d_str.size_bytes()};
  thrust::uninitialized_fill(
    rmm::exec_policy_nosync(stream), indices.begin(), indices.end(), row_value);
  return cudf::strings::detail::make_strings_column(indices.begin(), indices.end(), stream, mr);
}

template <>
std::unique_ptr<cudf::column> column_from_scalar_dispatch::operator()<cudf::dictionary32>(
  scalar const&, size_type, rmm::cuda_stream_view, rmm::device_async_resource_ref) const
{
  CUDF_FAIL("dictionary not supported when creating from scalar");
}

template <>
std::unique_ptr<cudf::column> column_from_scalar_dispatch::operator()<cudf::list_view>(
  scalar const& value,
  size_type size,
  rmm::cuda_stream_view stream,
  rmm::device_async_resource_ref mr) const
{
  auto lv = static_cast<list_scalar const*>(&value);
  return lists::detail::make_lists_column_from_scalar(*lv, size, stream, mr);
}

template <>
std::unique_ptr<cudf::column> column_from_scalar_dispatch::operator()<cudf::struct_view>(
  scalar const& value,
  size_type size,
  rmm::cuda_stream_view stream,
  rmm::device_async_resource_ref mr) const
{
  if (size == 0) CUDF_FAIL("0-length struct column is unsupported.");
  auto& ss  = static_cast<scalar_type_t<cudf::struct_view> const&>(value);
  auto iter = thrust::make_constant_iterator(0);

  auto children =
    detail::gather(ss.view(), iter, iter + size, out_of_bounds_policy::NULLIFY, stream, mr);
  auto const is_valid = ss.is_valid(stream);
  return make_structs_column(size,
                             std::move(children->release()),
                             is_valid ? 0 : size,
                             is_valid
                               ? rmm::device_buffer{}
                               : detail::create_null_mask(size, mask_state::ALL_NULL, stream, mr),
                             stream,
                             mr);
}

}  // anonymous namespace

std::unique_ptr<column> make_column_from_scalar(scalar const& s,
                                                size_type size,
                                                rmm::cuda_stream_view stream,
                                                rmm::device_async_resource_ref mr)
{
  return type_dispatcher(s.type(), column_from_scalar_dispatch{}, s, size, stream, mr);
}

}  // namespace cudf
