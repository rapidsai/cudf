/*
 * Copyright (c) 2019, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <cudf/column/column.hpp>
#include <cudf/column/column_view.hpp>
#include <cudf/null_mask.hpp>
#include <cudf/utilities/bit.hpp>
#include <cudf/utilities/traits.hpp>
#include <cudf/utilities/type_dispatcher.hpp>
#include <cudf/detail/null_mask.hpp>
#include <cudf/strings/copying.hpp>
#include <cudf/strings/detail/concatenate.hpp>
#include <cudf/copying.hpp>

#include <rmm/device_buffer.hpp>
#include <rmm/mr/device_memory_resource.hpp>

#include <algorithm>
#include <numeric>
#include <vector>

namespace cudf {

// Copy constructor
column::column(column const &other)
    : _type{other._type},
      _size{other._size},
      _data{other._data},
      _null_mask{other._null_mask},
      _null_count{other._null_count} {
  _children.reserve(other.num_children());
  for (auto const &c : other._children) {
    _children.emplace_back(std::make_unique<column>(*c));
  }
}

// Copy ctor w/ explicit stream/mr
column::column(column const &other, hipStream_t stream,
               rmm::mr::device_memory_resource *mr)
    : _type{other._type},
      _size{other._size},
      _data{other._data, stream, mr},
      _null_mask{other._null_mask, stream, mr},
      _null_count{other._null_count} {
  _children.reserve(other.num_children());
  for (auto const &c : other._children) {
    _children.emplace_back(std::make_unique<column>(*c, stream, mr));
  }
}

// Move constructor
column::column(column &&other) noexcept
    : _type{other._type},
      _size{other._size},
      _data{std::move(other._data)},
      _null_mask{std::move(other._null_mask)},
      _null_count{other._null_count},
      _children{std::move(other._children)} {
  other._size = 0;
  other._null_count = 0;
  other._type = data_type{EMPTY};
}

// Release contents
column::contents column::release() noexcept {
  _size = 0;
  _null_count = 0;
  _type = data_type{EMPTY};
  return column::contents{
      std::make_unique<rmm::device_buffer>(std::move(_data)),
      std::make_unique<rmm::device_buffer>(std::move(_null_mask)),
      std::move(_children)};
}

// Create immutable view
column_view column::view() const {
  // Create views of children
  std::vector<column_view> child_views;
  child_views.reserve(_children.size());
  for (auto const &c : _children) {
    child_views.emplace_back(*c);
  }

  return column_view{
      type(),       size(),
      _data.data(), static_cast<bitmask_type const *>(_null_mask.data()),
      null_count(), 0,
      child_views};
}

// Create mutable view
mutable_column_view column::mutable_view() {
  // create views of children
  std::vector<mutable_column_view> child_views;
  child_views.reserve(_children.size());
  for (auto const &c : _children) {
    child_views.emplace_back(*c);
  }

  // Store the old null count
  auto current_null_count = null_count();

  // The elements of a column could be changed through a `mutable_column_view`,
  // therefore the existing `null_count` is no longer valid. Reset it to
  // `UNKNOWN_NULL_COUNT` forcing it to be recomputed on the next invocation of
  // `null_count()`.
  set_null_count(cudf::UNKNOWN_NULL_COUNT);

  return mutable_column_view{type(),
                             size(),
                             _data.data(),
                             static_cast<bitmask_type *>(_null_mask.data()),
                             current_null_count,
                             0,
                             child_views};
}

// If the null count is known, return it. Else, compute and return it
size_type column::null_count() const {
  if (_null_count <= cudf::UNKNOWN_NULL_COUNT) {
    _null_count = cudf::count_unset_bits(
        static_cast<bitmask_type const *>(_null_mask.data()), 0, size());
  }
  return _null_count;
}

void column::set_null_mask(rmm::device_buffer&& new_null_mask,
                   size_type new_null_count) {
  if(new_null_count > 0){
    CUDF_EXPECTS(new_null_mask.size() >=
                   cudf::bitmask_allocation_size_bytes(this->size()),
                 "Column with null values must be nullable and the null mask \
                  buffer size should match the size of the column.");
    }
    _null_mask = std::move(new_null_mask);  // move
    _null_count = new_null_count;
}

void column::set_null_mask(rmm::device_buffer const& new_null_mask,
                   size_type new_null_count) {
  if(new_null_count > 0){
    CUDF_EXPECTS(new_null_mask.size() >=
                   cudf::bitmask_allocation_size_bytes(this->size()),
                 "Column with null values must be nullable and the null mask \
                  buffer size should match the size of the column.");
    }
    _null_mask = new_null_mask;  // copy
    _null_count = new_null_count;
}

void column::set_null_count(size_type new_null_count) {
  if (new_null_count > 0) {
    CUDF_EXPECTS(nullable(), "Invalid null count.");
  }
  _null_count = new_null_count;
}

struct create_column_from_view {
  cudf::column_view view;
  hipStream_t stream;
  rmm::mr::device_memory_resource *mr;

 template <typename ColumnType,
           std::enable_if_t<std::is_same<ColumnType, cudf::string_view>::value>* = nullptr>
 std::unique_ptr<column> operator()() {
   cudf::strings_column_view sview(view);
   return cudf::strings::detail::slice(sview, 0, view.size(), 1, stream, mr);
 }

 template <typename ColumnType,
           std::enable_if_t<cudf::is_fixed_width<ColumnType>()>* = nullptr>
 std::unique_ptr<column> operator()() {

   std::vector<std::unique_ptr<column>> children;
   for (size_type i = 0; i < view.num_children(); ++i) {
     children.emplace_back(std::make_unique<column>(view.child(i), stream, mr));
   }

   return std::make_unique<column>(view.type(), view.size(),
       rmm::device_buffer{
       static_cast<const char*>(view.head()) +
       (view.offset() * cudf::size_of(view.type())),
       view.size() * cudf::size_of(view.type()), stream, mr},
       cudf::copy_bitmask(view, stream, mr),
       view.null_count(), std::move(children));
 }

};

struct create_column_from_view_vector {
  std::vector<cudf::column_view> views;
  hipStream_t stream;
  rmm::mr::device_memory_resource *mr;

 template <typename ColumnType,
           std::enable_if_t<std::is_same<ColumnType, cudf::string_view>::value>* = nullptr>
 std::unique_ptr<column> operator()() {
   std::vector<cudf::strings_column_view> sviews;
   sviews.reserve(views.size());
   for (auto &v : views) { sviews.emplace_back(v); }

   auto col = cudf::strings::detail::concatenate(sviews, mr, stream);

   //If concatenated string column is nullable, proceed to calculate it
   if (col->nullable()) {
     cudf::detail::concatenate_masks(views,
         (col->mutable_view()).null_mask(), stream);
   }

   return col;
 }

 template <typename ColumnType,
           std::enable_if_t<cudf::is_fixed_width<ColumnType>()>* = nullptr>
 std::unique_ptr<column> operator()() {

   auto type = views.front().type();
   size_type total_element_count =
     std::accumulate(views.begin(), views.end(), 0,
         [](auto accumulator, auto const& v) { return accumulator + v.size(); });

   bool has_nulls = std::any_of(views.begin(), views.end(),
                      [](const column_view col) { return col.has_nulls(); });
   using mask_policy = cudf::experimental::mask_allocation_policy;

   mask_policy policy{mask_policy::NEVER};
   if (has_nulls) { policy = mask_policy::ALWAYS; }

   auto col = cudf::experimental::allocate_like(views.front(),
       total_element_count, policy, mr);

   auto m_view = col->mutable_view();
   auto count = 0;
   // TODO replace loop with a single kernel https://github.com/rapidsai/cudf/issues/2881
   for (auto &v : views) {
     thrust::copy(rmm::exec_policy()->on(stream),
         v.begin<ColumnType>(),
         v.end<ColumnType>(),
         m_view.begin<ColumnType>() + count);
     count += v.size();
   }

   //If concatenated column is nullable, proceed to calculate it
   if (col->nullable()) {
     cudf::detail::concatenate_masks(views,
         (col->mutable_view()).null_mask(), stream);
   }

   return col;
 }

};

// Copy from a view
column::column(column_view view, hipStream_t stream,
               rmm::mr::device_memory_resource *mr) :
  // Move is needed here because the dereference operator of unique_ptr returns
  // an lvalue reference, which would otherwise dispatch to the copy constructor
  column{std::move(*experimental::type_dispatcher(view.type(),
                    create_column_from_view{view, stream, mr}))} {}

// Concatenates the elements from a vector of column_views
std::unique_ptr<column>
concatenate(std::vector<column_view> const& columns_to_concat,
            rmm::mr::device_memory_resource *mr, hipStream_t stream) {
  if (columns_to_concat.empty()) { return std::make_unique<column>(); }

  data_type type = columns_to_concat.front().type();
  CUDF_EXPECTS(std::all_of(columns_to_concat.begin(), columns_to_concat.end(),
        [type](auto const& c) { return c.type() == type; }),
      "Type mismatch in columns to concatenate.");
  return cudf::experimental::type_dispatcher(type,
      create_column_from_view_vector{columns_to_concat, stream, mr});
}

}  // namespace cudf
