#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2020-2021, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <cudf/ast/detail/transform.cuh>
#include <cudf/ast/linearizer.hpp>
#include <cudf/ast/operators.hpp>
#include <cudf/ast/transform.hpp>
#include <cudf/column/column_device_view.cuh>
#include <cudf/column/column_factories.hpp>
#include <cudf/detail/nvtx/ranges.hpp>
#include <cudf/detail/utilities/cuda.cuh>
#include <cudf/scalar/scalar_device_view.cuh>
#include <cudf/table/table_device_view.cuh>
#include <cudf/table/table_view.hpp>
#include <cudf/types.hpp>
#include <cudf/utilities/error.hpp>
#include <cudf/utilities/traits.hpp>
#include <cudf/utilities/type_dispatcher.hpp>

#include <rmm/cuda_stream_view.hpp>
#include <rmm/device_buffer.hpp>
#include <rmm/mr/device/device_memory_resource.hpp>

#include <algorithm>
#include <functional>
#include <iterator>
#include <type_traits>

namespace cudf {
namespace ast {
namespace detail {

/**
 * @brief Kernel for evaluating an expression on a table to produce a new column.
 *
 * This evaluates an expression over a table to produce a new column. Also called an n-ary
 * transform.
 *
 * @tparam block_size
 * @param table The table device view used for evaluation.
 * @param literals Array of literal values used for evaluation.
 * @param output_column The output column where results are stored.
 * @param data_references Array of data references.
 * @param operators Array of operators to perform.
 * @param operator_source_indices Array of source indices for the operators.
 * @param num_operators Number of operators.
 * @param num_intermediates Number of intermediates, used to allocate a portion of shared memory to
 * each thread.
 */
template <cudf::size_type max_block_size>
__launch_bounds__(max_block_size) __global__
  void compute_column_kernel(table_device_view const table,
                             const cudf::detail::fixed_width_scalar_device_view_base* literals,
                             mutable_column_device_view output_column,
                             const detail::device_data_reference* data_references,
                             const ast_operator* operators,
                             const cudf::size_type* operator_source_indices,
                             cudf::size_type num_operators,
                             cudf::size_type num_intermediates)
{
  extern __shared__ std::int64_t intermediate_storage[];
  auto thread_intermediate_storage = &intermediate_storage[threadIdx.x * num_intermediates];
  auto const start_idx             = cudf::size_type(threadIdx.x + blockIdx.x * blockDim.x);
  auto const stride                = cudf::size_type(blockDim.x * gridDim.x);
  auto const num_rows              = table.num_rows();
  auto const evaluator =
    cudf::ast::detail::row_evaluator(table, literals, thread_intermediate_storage, &output_column);

  for (cudf::size_type row_index = start_idx; row_index < num_rows; row_index += stride) {
    evaluate_row_expression(
      evaluator, data_references, operators, operator_source_indices, num_operators, row_index);
  }
}

std::unique_ptr<column> compute_column(table_view const table,
                                       expression const& expr,
                                       rmm::cuda_stream_view stream,
                                       rmm::mr::device_memory_resource* mr)
{
  // Linearize the AST
  auto const expr_linearizer         = linearizer(expr, table);
  auto const data_references         = expr_linearizer.data_references();
  auto const literals                = expr_linearizer.literals();
  auto const operators               = expr_linearizer.operators();
  auto const num_operators           = cudf::size_type(operators.size());
  auto const operator_source_indices = expr_linearizer.operator_source_indices();
  auto const expr_data_type          = expr_linearizer.root_data_type();

  // Create ast_plan and device buffer
  auto plan = ast_plan();
  plan.add_to_plan(data_references);
  plan.add_to_plan(literals);
  plan.add_to_plan(operators);
  plan.add_to_plan(operator_source_indices);
  auto const host_data_buffer = plan.get_host_data_buffer();
  auto const buffer_offsets   = plan.get_offsets();
  auto const buffer_size      = host_data_buffer.second;
  auto device_data_buffer =
    rmm::device_buffer(host_data_buffer.first.get(), buffer_size, stream, mr);
  // To reduce overhead, we don't call a stream sync here.
  // The stream is synced later when the table_device_view is created.

  // Create device pointers to components of plan
  auto const device_data_buffer_ptr = static_cast<const char*>(device_data_buffer.data());
  auto const device_data_references = reinterpret_cast<const detail::device_data_reference*>(
    device_data_buffer_ptr + buffer_offsets[0]);
  auto const device_literals =
    reinterpret_cast<const cudf::detail::fixed_width_scalar_device_view_base*>(
      device_data_buffer_ptr + buffer_offsets[1]);
  auto const device_operators =
    reinterpret_cast<const ast_operator*>(device_data_buffer_ptr + buffer_offsets[2]);
  auto const device_operator_source_indices =
    reinterpret_cast<const cudf::size_type*>(device_data_buffer_ptr + buffer_offsets[3]);

  // Create table device view
  auto table_device         = table_device_view::create(table, stream);
  auto const table_num_rows = table.num_rows();

  // Prepare output column
  auto output_column = cudf::make_fixed_width_column(
    expr_data_type, table_num_rows, mask_state::UNALLOCATED, stream, mr);
  auto mutable_output_device =
    cudf::mutable_column_device_view::create(output_column->mutable_view(), stream);

  // Configure kernel parameters
  auto const num_intermediates     = expr_linearizer.intermediate_count();
  auto const shmem_size_per_thread = static_cast<int>(sizeof(std::int64_t) * num_intermediates);
  int device_id;
  CUDA_TRY(hipGetDevice(&device_id));
  int shmem_limit_per_block;
  CUDA_TRY(
    hipDeviceGetAttribute(&shmem_limit_per_block, hipDeviceAttributeMaxSharedMemoryPerBlock, device_id));
  auto constexpr MAX_BLOCK_SIZE = 128;
  auto const block_size =
    shmem_size_per_thread != 0
      ? std::min(MAX_BLOCK_SIZE, shmem_limit_per_block / shmem_size_per_thread)
      : MAX_BLOCK_SIZE;
  auto const config               = cudf::detail::grid_1d{table_num_rows, block_size};
  auto const shmem_size_per_block = shmem_size_per_thread * config.num_threads_per_block;

  // Execute the kernel
  cudf::ast::detail::compute_column_kernel<MAX_BLOCK_SIZE>
    <<<config.num_blocks, config.num_threads_per_block, shmem_size_per_block, stream.value()>>>(
      *table_device,
      device_literals,
      *mutable_output_device,
      device_data_references,
      device_operators,
      device_operator_source_indices,
      num_operators,
      num_intermediates);
  CHECK_CUDA(stream.value());
  return output_column;
}

}  // namespace detail

std::unique_ptr<column> compute_column(table_view const table,
                                       expression const& expr,
                                       rmm::mr::device_memory_resource* mr)
{
  CUDF_FUNC_RANGE();
  return detail::compute_column(table, expr, rmm::cuda_stream_default, mr);
}

}  // namespace ast

}  // namespace cudf
