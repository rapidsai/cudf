#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2020-2021, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <cudf/ast/detail/transform.cuh>
#include <cudf/ast/nodes.hpp>
#include <cudf/ast/operators.hpp>
#include <cudf/ast/transform.hpp>
#include <cudf/column/column_device_view.cuh>
#include <cudf/column/column_factories.hpp>
#include <cudf/detail/nvtx/ranges.hpp>
#include <cudf/detail/utilities/cuda.cuh>
#include <cudf/scalar/scalar_device_view.cuh>
#include <cudf/table/table_device_view.cuh>
#include <cudf/table/table_view.hpp>
#include <cudf/types.hpp>
#include <cudf/utilities/error.hpp>
#include <cudf/utilities/traits.hpp>
#include <cudf/utilities/type_dispatcher.hpp>

#include <rmm/cuda_stream_view.hpp>
#include <rmm/device_buffer.hpp>
#include <rmm/mr/device/device_memory_resource.hpp>

#include <algorithm>
#include <functional>
#include <iterator>
#include <type_traits>

namespace cudf {
namespace ast {
namespace detail {

/**
 * @brief Kernel for evaluating an expression on a table to produce a new column.
 *
 * This evaluates an expression over a table to produce a new column. Also called an n-ary
 * transform.
 *
 * @tparam block_size
 * @param table The table device view used for evaluation.
 * @param literals Array of literal values used for evaluation.
 * @param output_column The output column where results are stored.
 * @param data_references Array of data references.
 * @param operators Array of operators to perform.
 * @param operator_source_indices Array of source indices for the operators.
 * @param num_operators Number of operators.
 * @param num_intermediates Number of intermediates, used to allocate a portion of shared memory to
 * each thread.
 */
template <cudf::size_type max_block_size>
__launch_bounds__(max_block_size) __global__ void compute_column_kernel(
  table_device_view const table,
  device_span<const cudf::detail::fixed_width_scalar_device_view_base> literals,
  mutable_column_device_view output_column,
  device_span<const detail::device_data_reference> data_references,
  device_span<const ast_operator> operators,
  device_span<const cudf::size_type> operator_source_indices,
  cudf::size_type num_intermediates)
{
  extern __shared__ std::int64_t intermediate_storage[];
  auto thread_intermediate_storage = &intermediate_storage[threadIdx.x * num_intermediates];
  auto const start_idx = static_cast<cudf::size_type>(threadIdx.x + blockIdx.x * blockDim.x);
  auto const stride    = static_cast<cudf::size_type>(blockDim.x * gridDim.x);
  auto const evaluator =
    cudf::ast::detail::row_evaluator(table, literals, thread_intermediate_storage, &output_column);

  for (cudf::size_type row_index = start_idx; row_index < table.num_rows(); row_index += stride) {
    evaluate_row_expression(
      evaluator, data_references, operators, operator_source_indices, row_index);
  }
}

std::unique_ptr<column> compute_column(table_view const table,
                                       expression const& expr,
                                       rmm::cuda_stream_view stream,
                                       rmm::mr::device_memory_resource* mr)
{
  auto const expr_linearizer = linearizer(expr, table);                // Linearize the AST
  auto const plan            = ast_plan{expr_linearizer, stream, mr};  // Create ast_plan

  // Create table device view
  auto table_device         = table_device_view::create(table, stream);
  auto const table_num_rows = table.num_rows();

  // Prepare output column
  auto output_column = cudf::make_fixed_width_column(
    expr_linearizer.root_data_type(), table_num_rows, mask_state::UNALLOCATED, stream, mr);
  auto mutable_output_device =
    cudf::mutable_column_device_view::create(output_column->mutable_view(), stream);

  // Configure kernel parameters
  auto const num_intermediates     = expr_linearizer.intermediate_count();
  auto const shmem_size_per_thread = static_cast<int>(sizeof(std::int64_t) * num_intermediates);
  int device_id;
  CUDA_TRY(hipGetDevice(&device_id));
  int shmem_limit_per_block;
  CUDA_TRY(
    hipDeviceGetAttribute(&shmem_limit_per_block, hipDeviceAttributeMaxSharedMemoryPerBlock, device_id));
  auto constexpr MAX_BLOCK_SIZE = 128;
  auto const block_size =
    shmem_size_per_thread != 0
      ? std::min(MAX_BLOCK_SIZE, shmem_limit_per_block / shmem_size_per_thread)
      : MAX_BLOCK_SIZE;
  auto const config               = cudf::detail::grid_1d{table_num_rows, block_size};
  auto const shmem_size_per_block = shmem_size_per_thread * config.num_threads_per_block;

  // Execute the kernel
  cudf::ast::detail::compute_column_kernel<MAX_BLOCK_SIZE>
    <<<config.num_blocks, config.num_threads_per_block, shmem_size_per_block, stream.value()>>>(
      *table_device,
      plan._device_literals,
      *mutable_output_device,
      plan._device_data_references,
      plan._device_operators,
      plan._device_operator_source_indices,
      num_intermediates);
  CHECK_CUDA(stream.value());
  return output_column;
}

}  // namespace detail

std::unique_ptr<column> compute_column(table_view const table,
                                       expression const& expr,
                                       rmm::mr::device_memory_resource* mr)
{
  CUDF_FUNC_RANGE();
  return detail::compute_column(table, expr, rmm::cuda_stream_default, mr);
}

}  // namespace ast

}  // namespace cudf
