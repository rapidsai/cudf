#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2020-2021, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <cudf/ast/detail/expression_evaluator.cuh>
#include <cudf/ast/detail/expression_parser.hpp>
#include <cudf/ast/nodes.hpp>
#include <cudf/ast/operators.hpp>
#include <cudf/ast/transform.hpp>
#include <cudf/column/column_device_view.cuh>
#include <cudf/column/column_factories.hpp>
#include <cudf/detail/nvtx/ranges.hpp>
#include <cudf/detail/utilities/cuda.cuh>
#include <cudf/scalar/scalar_device_view.cuh>
#include <cudf/table/table_device_view.cuh>
#include <cudf/table/table_view.hpp>
#include <cudf/types.hpp>
#include <cudf/utilities/error.hpp>
#include <cudf/utilities/type_dispatcher.hpp>

#include <rmm/cuda_stream_view.hpp>
#include <rmm/mr/device/device_memory_resource.hpp>

namespace cudf {
namespace ast {
namespace detail {

/**
 * @brief Kernel for evaluating an expression on a table to produce a new column.
 *
 * This evaluates an expression over a table to produce a new column. Also called an n-ary
 * transform.
 *
 * @tparam max_block_size The size of the thread block, used to set launch
 * bounds and minimize register usage.
 * @tparam has_nulls whether or not the output column may contain nulls.
 *
 * @param table The table device view used for evaluation.
 * @param device_expression_data Container of device data required to evaluate the desired
 * expression.
 * @param output_column The destination for the results of evaluating the expression.
 */
template <cudf::size_type max_block_size, bool has_nulls>
__launch_bounds__(max_block_size) __global__
  void compute_column_kernel(table_device_view const table,
                             ast::detail::expression_device_view device_expression_data,
                             mutable_column_device_view output_column)
{
  // The (required) extern storage of the shared memory array leads to
  // conflicting declarations between different templates. The easiest
  // workaround is to declare an arbitrary (here char) array type then cast it
  // after the fact to the appropriate type.
  extern __shared__ char raw_intermediate_storage[];
  IntermediateDataType<has_nulls>* intermediate_storage =
    reinterpret_cast<IntermediateDataType<has_nulls>*>(raw_intermediate_storage);

  auto thread_intermediate_storage =
    &intermediate_storage[threadIdx.x * device_expression_data.num_intermediates];
  auto const start_idx = static_cast<cudf::size_type>(threadIdx.x + blockIdx.x * blockDim.x);
  auto const stride    = static_cast<cudf::size_type>(blockDim.x * gridDim.x);
  auto evaluator       = cudf::ast::detail::expression_evaluator<has_nulls>(
    table, device_expression_data, thread_intermediate_storage);

  for (cudf::size_type row_index = start_idx; row_index < table.num_rows(); row_index += stride) {
    auto output_dest = mutable_column_expression_result<has_nulls>(output_column);
    evaluator.evaluate(output_dest, row_index);
  }
}

std::unique_ptr<column> compute_column(table_view const table,
                                       expression const& expr,
                                       rmm::cuda_stream_view stream,
                                       rmm::mr::device_memory_resource* mr)
{
  // Prepare output column. Whether or not the output column is nullable is
  // determined by whether any of the columns in the input table are nullable.
  // If none of the input columns actually contain nulls, we can still use the
  // non-nullable version of the expression evaluation code path for
  // performance, so we capture that information as well.
  auto const nullable  = cudf::nullable(table);
  auto const has_nulls = nullable && cudf::has_nulls(table);

  auto const parser = ast::detail::expression_parser{expr, table, has_nulls, stream, mr};

  auto const output_column_mask_state =
    nullable ? (has_nulls ? mask_state::UNINITIALIZED : mask_state::ALL_VALID)
             : mask_state::UNALLOCATED;

  auto output_column = cudf::make_fixed_width_column(
    parser.output_type(), table.num_rows(), output_column_mask_state, stream, mr);
  auto mutable_output_device =
    cudf::mutable_column_device_view::create(output_column->mutable_view(), stream);

  // Configure kernel parameters
  auto const& device_expression_data = parser.device_expression_data;
  int device_id;
  CUDA_TRY(hipGetDevice(&device_id));
  int shmem_limit_per_block;
  CUDA_TRY(
    hipDeviceGetAttribute(&shmem_limit_per_block, hipDeviceAttributeMaxSharedMemoryPerBlock, device_id));
  auto constexpr MAX_BLOCK_SIZE = 128;
  auto const block_size =
    device_expression_data.shmem_per_thread != 0
      ? std::min(MAX_BLOCK_SIZE, shmem_limit_per_block / device_expression_data.shmem_per_thread)
      : MAX_BLOCK_SIZE;
  auto const config = cudf::detail::grid_1d{table.num_rows(), block_size};
  auto const shmem_per_block =
    device_expression_data.shmem_per_thread * config.num_threads_per_block;

  // Execute the kernel
  auto table_device = table_device_view::create(table, stream);
  if (has_nulls) {
    cudf::ast::detail::compute_column_kernel<MAX_BLOCK_SIZE, true>
      <<<config.num_blocks, config.num_threads_per_block, shmem_per_block, stream.value()>>>(
        *table_device, device_expression_data, *mutable_output_device);
  } else {
    cudf::ast::detail::compute_column_kernel<MAX_BLOCK_SIZE, false>
      <<<config.num_blocks, config.num_threads_per_block, shmem_per_block, stream.value()>>>(
        *table_device, device_expression_data, *mutable_output_device);
  }
  CHECK_CUDA(stream.value());
  return output_column;
}

}  // namespace detail

std::unique_ptr<column> compute_column(table_view const table,
                                       expression const& expr,
                                       rmm::mr::device_memory_resource* mr)
{
  CUDF_FUNC_RANGE();
  return detail::compute_column(table, expr, rmm::cuda_stream_default, mr);
}

}  // namespace ast

}  // namespace cudf
