/*
 * Copyright (c) 2024, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "arrow_utilities.hpp"
#include "decimal_conversion_utilities.cuh"

#include <cudf/column/column.hpp>
#include <cudf/column/column_view.hpp>
#include <cudf/detail/interop.hpp>
#include <cudf/detail/iterator.cuh>
#include <cudf/detail/null_mask.hpp>
#include <cudf/detail/nvtx/ranges.hpp>
#include <cudf/dictionary/dictionary_column_view.hpp>
#include <cudf/interop.hpp>
#include <cudf/lists/lists_column_view.hpp>
#include <cudf/null_mask.hpp>
#include <cudf/strings/strings_column_view.hpp>
#include <cudf/table/table_view.hpp>
#include <cudf/types.hpp>
#include <cudf/utilities/memory_resource.hpp>
#include <cudf/utilities/traits.hpp>
#include <cudf/utilities/type_dispatcher.hpp>

#include <rmm/cuda_stream_view.hpp>
#include <rmm/device_scalar.hpp>
#include <rmm/exec_policy.hpp>

#include <thrust/for_each.h>
#include <thrust/iterator/counting_iterator.h>

#include <nanoarrow/nanoarrow.h>
#include <nanoarrow/nanoarrow.hpp>
#include <nanoarrow/nanoarrow_device.h>

namespace cudf {
namespace detail {
namespace {

template <typename T>
void device_buffer_finalize(ArrowBufferAllocator* allocator, uint8_t*, int64_t)
{
  auto* unique_buffer = reinterpret_cast<std::unique_ptr<T>*>(allocator->private_data);
  delete unique_buffer;
}

template <typename>
struct is_device_scalar : public std::false_type {};

template <typename T>
struct is_device_scalar<rmm::device_scalar<T>> : public std::true_type {};

template <typename>
struct is_device_uvector : public std::false_type {};

template <typename T>
struct is_device_uvector<rmm::device_uvector<T>> : public std::true_type {};

template <typename T>
int set_buffer(std::unique_ptr<T> device_buf, int64_t i, ArrowArray* out)
{
  ArrowBuffer* buf = ArrowArrayBuffer(out, i);
  auto ptr         = reinterpret_cast<uint8_t*>(device_buf->data());
  buf->size_bytes  = [&] {
    if constexpr (is_device_scalar<T>::value) {
      return sizeof(typename T::value_type);
    } else if constexpr (is_device_uvector<T>::value) {
      return sizeof(typename T::value_type) * device_buf->size();
    } else {
      return device_buf->size();
    }
  }();
  // we make a new unique_ptr and move to it in case there was a custom deleter
  NANOARROW_RETURN_NOT_OK(
    ArrowBufferSetAllocator(buf,
                            ArrowBufferDeallocator(&device_buffer_finalize<T>,
                                                   new std::unique_ptr<T>(std::move(device_buf)))));
  buf->data = ptr;
  return NANOARROW_OK;
}

struct dispatch_to_arrow_device {
  template <typename T, CUDF_ENABLE_IF(not is_rep_layout_compatible<T>())>
  int operator()(cudf::column&&, rmm::cuda_stream_view, rmm::device_async_resource_ref, ArrowArray*)
  {
    CUDF_FAIL("Unsupported type for to_arrow_device", cudf::data_type_error);
  }

  template <typename T, CUDF_ENABLE_IF(is_rep_layout_compatible<T>())>
  int operator()(cudf::column&& column,
                 rmm::cuda_stream_view stream,
                 rmm::device_async_resource_ref mr,
                 ArrowArray* out)
  {
    nanoarrow::UniqueArray tmp;

    auto const storage_type = id_to_arrow_storage_type(column.type().id());
    NANOARROW_RETURN_NOT_OK(initialize_array(tmp.get(), storage_type, column));

    auto contents = column.release();
    NANOARROW_RETURN_NOT_OK(set_contents(contents, tmp.get()));

    ArrowArrayMove(tmp.get(), out);
    return NANOARROW_OK;
  }

  int set_null_mask(column::contents& contents, ArrowArray* out)
  {
    if (contents.null_mask) {
      NANOARROW_RETURN_NOT_OK(set_buffer(std::move(contents.null_mask), validity_buffer_idx, out));
    }
    return NANOARROW_OK;
  }

  int set_contents(column::contents& contents, ArrowArray* out)
  {
    NANOARROW_RETURN_NOT_OK(set_null_mask(contents, out));
    NANOARROW_RETURN_NOT_OK(set_buffer(std::move(contents.data), fixed_width_data_buffer_idx, out));
    return NANOARROW_OK;
  }
};

template <typename DeviceType>
int construct_decimals(cudf::column_view input,
                       rmm::cuda_stream_view stream,
                       rmm::device_async_resource_ref mr,
                       ArrowArray* out)
{
  nanoarrow::UniqueArray tmp;
  NANOARROW_RETURN_NOT_OK(initialize_array(tmp.get(), NANOARROW_TYPE_DECIMAL128, input));

  auto buf = detail::convert_decimals_to_decimal128<DeviceType>(input, stream, mr);
  // Synchronize stream here to ensure the decimal128 buffer is ready.
  stream.synchronize();
  NANOARROW_RETURN_NOT_OK(set_buffer(std::move(buf), fixed_width_data_buffer_idx, tmp.get()));

  ArrowArrayMove(tmp.get(), out);
  return NANOARROW_OK;
}

template <>
int dispatch_to_arrow_device::operator()<numeric::decimal32>(cudf::column&& column,
                                                             rmm::cuda_stream_view stream,
                                                             rmm::device_async_resource_ref mr,
                                                             ArrowArray* out)
{
  using DeviceType = int32_t;
  NANOARROW_RETURN_NOT_OK(construct_decimals<DeviceType>(column.view(), stream, mr, out));
  auto contents = column.release();
  NANOARROW_RETURN_NOT_OK(set_null_mask(contents, out));
  return NANOARROW_OK;
}

template <>
int dispatch_to_arrow_device::operator()<numeric::decimal64>(cudf::column&& column,
                                                             rmm::cuda_stream_view stream,
                                                             rmm::device_async_resource_ref mr,
                                                             ArrowArray* out)
{
  using DeviceType = int64_t;
  NANOARROW_RETURN_NOT_OK(construct_decimals<DeviceType>(column.view(), stream, mr, out));
  auto contents = column.release();
  NANOARROW_RETURN_NOT_OK(set_null_mask(contents, out));
  return NANOARROW_OK;
}

template <>
int dispatch_to_arrow_device::operator()<numeric::decimal128>(cudf::column&& column,
                                                              rmm::cuda_stream_view stream,
                                                              rmm::device_async_resource_ref mr,
                                                              ArrowArray* out)
{
  nanoarrow::UniqueArray tmp;
  NANOARROW_RETURN_NOT_OK(initialize_array(tmp.get(), NANOARROW_TYPE_DECIMAL128, column));
  auto contents = column.release();
  NANOARROW_RETURN_NOT_OK(set_contents(contents, tmp.get()));
  ArrowArrayMove(tmp.get(), out);
  return NANOARROW_OK;
}

template <>
int dispatch_to_arrow_device::operator()<bool>(cudf::column&& column,
                                               rmm::cuda_stream_view stream,
                                               rmm::device_async_resource_ref mr,
                                               ArrowArray* out)
{
  nanoarrow::UniqueArray tmp;
  NANOARROW_RETURN_NOT_OK(initialize_array(tmp.get(), NANOARROW_TYPE_BOOL, column));

  auto bitmask  = detail::bools_to_mask(column.view(), stream, mr);
  auto contents = column.release();
  NANOARROW_RETURN_NOT_OK(set_null_mask(contents, tmp.get()));
  NANOARROW_RETURN_NOT_OK(
    set_buffer(std::move(bitmask.first), fixed_width_data_buffer_idx, tmp.get()));

  ArrowArrayMove(tmp.get(), out);
  return NANOARROW_OK;
}

template <>
int dispatch_to_arrow_device::operator()<cudf::string_view>(cudf::column&& column,
                                                            rmm::cuda_stream_view stream,
                                                            rmm::device_async_resource_ref mr,
                                                            ArrowArray* out)
{
  ArrowType nanoarrow_type = NANOARROW_TYPE_STRING;
  if (column.num_children() > 0 &&
      column.child(cudf::strings_column_view::offsets_column_index).type().id() ==
        cudf::type_id::INT64) {
    nanoarrow_type = NANOARROW_TYPE_LARGE_STRING;
  }

  nanoarrow::UniqueArray tmp;
  NANOARROW_RETURN_NOT_OK(initialize_array(tmp.get(), nanoarrow_type, column));

  if (column.size() == 0) {
    // the scalar zero here is necessary because the spec for string arrays states
    // that the offsets buffer should contain "length + 1" signed integers. So in
    // the case of a 0 length string array, there should be exactly 1 value, zero,
    // in the offsets buffer. While some arrow implementations may accept a zero-sized
    // offsets buffer, best practices would be to allocate the buffer with the single value.
    if (nanoarrow_type == NANOARROW_TYPE_STRING) {
      auto zero = std::make_unique<rmm::device_scalar<int32_t>>(0, stream, mr);
      NANOARROW_RETURN_NOT_OK(set_buffer(std::move(zero), fixed_width_data_buffer_idx, tmp.get()));
    } else {
      auto zero = std::make_unique<rmm::device_scalar<int64_t>>(0, stream, mr);
      NANOARROW_RETURN_NOT_OK(set_buffer(std::move(zero), fixed_width_data_buffer_idx, tmp.get()));
    }

    ArrowArrayMove(tmp.get(), out);
    return NANOARROW_OK;
  }

  auto contents = column.release();
  NANOARROW_RETURN_NOT_OK(set_null_mask(contents, tmp.get()));

  auto offsets_contents =
    contents.children[cudf::strings_column_view::offsets_column_index]->release();
  NANOARROW_RETURN_NOT_OK(set_buffer(std::move(offsets_contents.data), 1, tmp.get()));
  NANOARROW_RETURN_NOT_OK(set_buffer(std::move(contents.data), 2, tmp.get()));

  ArrowArrayMove(tmp.get(), out);
  return NANOARROW_OK;
}

template <>
int dispatch_to_arrow_device::operator()<cudf::list_view>(cudf::column&& column,
                                                          rmm::cuda_stream_view stream,
                                                          rmm::device_async_resource_ref mr,
                                                          ArrowArray* out);

template <>
int dispatch_to_arrow_device::operator()<cudf::dictionary32>(cudf::column&& column,
                                                             rmm::cuda_stream_view stream,
                                                             rmm::device_async_resource_ref mr,
                                                             ArrowArray* out);

template <>
int dispatch_to_arrow_device::operator()<cudf::struct_view>(cudf::column&& column,
                                                            rmm::cuda_stream_view stream,
                                                            rmm::device_async_resource_ref mr,
                                                            ArrowArray* out)
{
  nanoarrow::UniqueArray tmp;
  NANOARROW_RETURN_NOT_OK(initialize_array(tmp.get(), NANOARROW_TYPE_STRUCT, column));
  NANOARROW_RETURN_NOT_OK(ArrowArrayAllocateChildren(tmp.get(), column.num_children()));

  auto contents = column.release();
  NANOARROW_RETURN_NOT_OK(set_null_mask(contents, tmp.get()));

  for (size_t i = 0; i < size_t(tmp->n_children); ++i) {
    ArrowArray* child_ptr = tmp->children[i];
    auto& child           = contents.children[i];
    NANOARROW_RETURN_NOT_OK(cudf::type_dispatcher(
      child->type(), dispatch_to_arrow_device{}, std::move(*child), stream, mr, child_ptr));
  }

  ArrowArrayMove(tmp.get(), out);
  return NANOARROW_OK;
}

template <>
int dispatch_to_arrow_device::operator()<cudf::list_view>(cudf::column&& column,
                                                          rmm::cuda_stream_view stream,
                                                          rmm::device_async_resource_ref mr,
                                                          ArrowArray* out)
{
  nanoarrow::UniqueArray tmp;
  NANOARROW_RETURN_NOT_OK(initialize_array(tmp.get(), NANOARROW_TYPE_LIST, column));
  NANOARROW_RETURN_NOT_OK(ArrowArrayAllocateChildren(tmp.get(), 1));

  auto contents = column.release();
  NANOARROW_RETURN_NOT_OK(set_null_mask(contents, tmp.get()));

  auto offsets_contents =
    contents.children[cudf::lists_column_view::offsets_column_index]->release();
  NANOARROW_RETURN_NOT_OK(set_buffer(std::move(offsets_contents.data), 1, tmp.get()));

  auto& child = contents.children[cudf::lists_column_view::child_column_index];
  NANOARROW_RETURN_NOT_OK(cudf::type_dispatcher(
    child->type(), dispatch_to_arrow_device{}, std::move(*child), stream, mr, tmp->children[0]));

  ArrowArrayMove(tmp.get(), out);
  return NANOARROW_OK;
}

template <>
int dispatch_to_arrow_device::operator()<cudf::dictionary32>(cudf::column&& column,
                                                             rmm::cuda_stream_view stream,
                                                             rmm::device_async_resource_ref mr,
                                                             ArrowArray* out)
{
  nanoarrow::UniqueArray tmp;
  NANOARROW_RETURN_NOT_OK(initialize_array(
    tmp.get(),
    id_to_arrow_type(column.child(cudf::dictionary_column_view::indices_column_index).type().id()),
    column));
  NANOARROW_RETURN_NOT_OK(ArrowArrayAllocateDictionary(tmp.get()));

  auto contents = column.release();
  NANOARROW_RETURN_NOT_OK(set_null_mask(contents, tmp.get()));

  auto indices_contents =
    contents.children[cudf::dictionary_column_view::indices_column_index]->release();
  NANOARROW_RETURN_NOT_OK(
    set_buffer(std::move(indices_contents.data), fixed_width_data_buffer_idx, tmp.get()));

  auto& keys = contents.children[cudf::dictionary_column_view::keys_column_index];
  NANOARROW_RETURN_NOT_OK(cudf::type_dispatcher(
    keys->type(), dispatch_to_arrow_device{}, std::move(*keys), stream, mr, tmp->dictionary));

  ArrowArrayMove(tmp.get(), out);
  return NANOARROW_OK;
}

struct dispatch_to_arrow_device_view {
  cudf::column_view column;
  rmm::cuda_stream_view stream;
  rmm::device_async_resource_ref mr;

  template <typename T, CUDF_ENABLE_IF(not is_rep_layout_compatible<T>())>
  int operator()(ArrowArray*) const
  {
    CUDF_FAIL("Unsupported type for to_arrow_device", cudf::data_type_error);
  }

  template <typename T, CUDF_ENABLE_IF(is_rep_layout_compatible<T>())>
  int operator()(ArrowArray* out) const
  {
    nanoarrow::UniqueArray tmp;

    auto const storage_type = id_to_arrow_storage_type(column.type().id());
    NANOARROW_RETURN_NOT_OK(initialize_array(tmp.get(), storage_type, column));
    NANOARROW_RETURN_NOT_OK(set_null_mask(column, tmp.get()));
    NANOARROW_RETURN_NOT_OK(set_view_to_buffer(column, tmp.get()));

    ArrowArrayMove(tmp.get(), out);
    return NANOARROW_OK;
  }

  int set_buffer_view(void const* in_ptr, size_t size, int64_t i, ArrowArray* out) const
  {
    ArrowBuffer* buf = ArrowArrayBuffer(out, i);
    buf->size_bytes  = size;

    // reset the deallocator to do nothing since this is a non-owning view
    NANOARROW_RETURN_NOT_OK(ArrowBufferSetAllocator(
      buf, ArrowBufferDeallocator([](ArrowBufferAllocator*, uint8_t*, int64_t) {}, nullptr)));

    buf->data = const_cast<uint8_t*>(reinterpret_cast<uint8_t const*>(in_ptr));
    return NANOARROW_OK;
  }

  int set_null_mask(column_view column, ArrowArray* out) const
  {
    if (column.nullable()) {
      NANOARROW_RETURN_NOT_OK(set_buffer_view(column.null_mask(),
                                              bitmask_allocation_size_bytes(column.size()),
                                              validity_buffer_idx,
                                              out));
    }
    return NANOARROW_OK;
  }

  int set_view_to_buffer(column_view column, ArrowArray* out) const
  {
    auto const type_size = cudf::size_of(column.type());
    return set_buffer_view(column.head<uint8_t>() + (type_size * column.offset()),
                           column.size() * type_size,
                           fixed_width_data_buffer_idx,
                           out);
  }
};

template <>
int dispatch_to_arrow_device_view::operator()<numeric::decimal32>(ArrowArray* out) const
{
  using DeviceType = int32_t;
  NANOARROW_RETURN_NOT_OK(construct_decimals<DeviceType>(column, stream, mr, out));
  NANOARROW_RETURN_NOT_OK(set_null_mask(column, out));
  return NANOARROW_OK;
}

template <>
int dispatch_to_arrow_device_view::operator()<numeric::decimal64>(ArrowArray* out) const
{
  using DeviceType = int64_t;
  NANOARROW_RETURN_NOT_OK(construct_decimals<DeviceType>(column, stream, mr, out));
  NANOARROW_RETURN_NOT_OK(set_null_mask(column, out));
  return NANOARROW_OK;
}

template <>
int dispatch_to_arrow_device_view::operator()<numeric::decimal128>(ArrowArray* out) const
{
  nanoarrow::UniqueArray tmp;

  NANOARROW_RETURN_NOT_OK(initialize_array(tmp.get(), NANOARROW_TYPE_DECIMAL128, column));
  NANOARROW_RETURN_NOT_OK(set_null_mask(column, tmp.get()));
  NANOARROW_RETURN_NOT_OK(set_view_to_buffer(column, tmp.get()));

  ArrowArrayMove(tmp.get(), out);
  return NANOARROW_OK;
}

template <>
int dispatch_to_arrow_device_view::operator()<bool>(ArrowArray* out) const
{
  nanoarrow::UniqueArray tmp;
  NANOARROW_RETURN_NOT_OK(initialize_array(tmp.get(), NANOARROW_TYPE_BOOL, column));

  auto bitmask = detail::bools_to_mask(column, stream, mr);
  NANOARROW_RETURN_NOT_OK(
    set_buffer(std::move(bitmask.first), fixed_width_data_buffer_idx, tmp.get()));
  NANOARROW_RETURN_NOT_OK(set_null_mask(column, tmp.get()));

  ArrowArrayMove(tmp.get(), out);
  return NANOARROW_OK;
}

template <>
int dispatch_to_arrow_device_view::operator()<cudf::string_view>(ArrowArray* out) const
{
  ArrowType nanoarrow_type = NANOARROW_TYPE_STRING;
  if (column.num_children() > 0 &&
      column.child(cudf::strings_column_view::offsets_column_index).type().id() ==
        cudf::type_id::INT64) {
    nanoarrow_type = NANOARROW_TYPE_LARGE_STRING;
  }

  nanoarrow::UniqueArray tmp;
  NANOARROW_RETURN_NOT_OK(initialize_array(tmp.get(), nanoarrow_type, column));

  if (column.size() == 0) {
    // https://github.com/rapidsai/cudf/pull/15047#discussion_r1546528552
    if (nanoarrow_type == NANOARROW_TYPE_LARGE_STRING) {
      auto zero = std::make_unique<rmm::device_scalar<int64_t>>(0, stream, mr);
      NANOARROW_RETURN_NOT_OK(set_buffer(std::move(zero), fixed_width_data_buffer_idx, tmp.get()));
    } else {
      auto zero = std::make_unique<rmm::device_scalar<int32_t>>(0, stream, mr);
      NANOARROW_RETURN_NOT_OK(set_buffer(std::move(zero), fixed_width_data_buffer_idx, tmp.get()));
    }

    ArrowArrayMove(tmp.get(), out);
    return NANOARROW_OK;
  }

  NANOARROW_RETURN_NOT_OK(set_null_mask(column, tmp.get()));

  auto const scv = cudf::strings_column_view(column);
  NANOARROW_RETURN_NOT_OK(set_view_to_buffer(scv.offsets(), tmp.get()));
  NANOARROW_RETURN_NOT_OK(
    set_buffer_view(scv.chars_begin(stream), scv.chars_size(stream), 2, tmp.get()));

  ArrowArrayMove(tmp.get(), out);
  return NANOARROW_OK;
}

template <>
int dispatch_to_arrow_device_view::operator()<cudf::list_view>(ArrowArray* out) const;

template <>
int dispatch_to_arrow_device_view::operator()<cudf::dictionary32>(ArrowArray* out) const;

template <>
int dispatch_to_arrow_device_view::operator()<cudf::struct_view>(ArrowArray* out) const
{
  nanoarrow::UniqueArray tmp;

  NANOARROW_RETURN_NOT_OK(initialize_array(tmp.get(), NANOARROW_TYPE_STRUCT, column));
  NANOARROW_RETURN_NOT_OK(ArrowArrayAllocateChildren(tmp.get(), column.num_children()));
  NANOARROW_RETURN_NOT_OK(set_null_mask(column, tmp.get()));

  for (size_t i = 0; i < size_t(tmp->n_children); ++i) {
    ArrowArray* child_ptr = tmp->children[i];
    auto const child      = column.child(i);
    NANOARROW_RETURN_NOT_OK(cudf::type_dispatcher(
      child.type(), dispatch_to_arrow_device_view{child, stream, mr}, child_ptr));
  }

  ArrowArrayMove(tmp.get(), out);
  return NANOARROW_OK;
}

template <>
int dispatch_to_arrow_device_view::operator()<cudf::list_view>(ArrowArray* out) const
{
  nanoarrow::UniqueArray tmp;

  NANOARROW_RETURN_NOT_OK(initialize_array(tmp.get(), NANOARROW_TYPE_LIST, column));
  NANOARROW_RETURN_NOT_OK(ArrowArrayAllocateChildren(tmp.get(), 1));
  NANOARROW_RETURN_NOT_OK(set_null_mask(column, tmp.get()));

  auto const lcv = cudf::lists_column_view(column);
  NANOARROW_RETURN_NOT_OK(set_view_to_buffer(lcv.offsets(), tmp.get()));

  auto child = lcv.child();
  NANOARROW_RETURN_NOT_OK(cudf::type_dispatcher(
    child.type(), dispatch_to_arrow_device_view{child, stream, mr}, tmp->children[0]));

  ArrowArrayMove(tmp.get(), out);
  return NANOARROW_OK;
}

template <>
int dispatch_to_arrow_device_view::operator()<cudf::dictionary32>(ArrowArray* out) const
{
  nanoarrow::UniqueArray tmp;

  NANOARROW_RETURN_NOT_OK(initialize_array(
    tmp.get(),
    id_to_arrow_type(column.child(cudf::dictionary_column_view::indices_column_index).type().id()),
    column));
  NANOARROW_RETURN_NOT_OK(ArrowArrayAllocateDictionary(tmp.get()));
  NANOARROW_RETURN_NOT_OK(set_null_mask(column, tmp.get()));

  auto const dcv = cudf::dictionary_column_view(column);
  NANOARROW_RETURN_NOT_OK(set_view_to_buffer(dcv.indices(), tmp.get()));

  auto keys = dcv.keys();
  NANOARROW_RETURN_NOT_OK(cudf::type_dispatcher(
    keys.type(), dispatch_to_arrow_device_view{keys, stream, mr}, tmp->dictionary));

  ArrowArrayMove(tmp.get(), out);
  return NANOARROW_OK;
}

struct ArrowDeviceArrayPrivateData {
  ArrowArray parent;
  hipEvent_t sync_event;
};

void ArrowDeviceArrayRelease(ArrowArray* array)
{
  auto private_data = reinterpret_cast<ArrowDeviceArrayPrivateData*>(array->private_data);
  RMM_ASSERT_CUDA_SUCCESS(hipEventDestroy(private_data->sync_event));
  ArrowArrayRelease(&private_data->parent);
  delete private_data;
  array->release = nullptr;
}

unique_device_array_t create_device_array(nanoarrow::UniqueArray&& out,
                                          rmm::cuda_stream_view stream)
{
  NANOARROW_THROW_NOT_OK(
    ArrowArrayFinishBuilding(out.get(), NANOARROW_VALIDATION_LEVEL_MINIMAL, nullptr));

  auto private_data = std::make_unique<detail::ArrowDeviceArrayPrivateData>();
  CUDF_CUDA_TRY(hipEventCreate(&private_data->sync_event));
  CUDF_CUDA_TRY(hipEventRecord(private_data->sync_event, stream.value()));

  ArrowArrayMove(out.get(), &private_data->parent);
  unique_device_array_t result(new ArrowDeviceArray, [](ArrowDeviceArray* arr) {
    if (arr->array.release != nullptr) { ArrowArrayRelease(&arr->array); }
    delete arr;
  });
  result->device_id          = rmm::get_current_cuda_device().value();
  result->device_type        = ARROW_DEVICE_CUDA;
  result->sync_event         = &private_data->sync_event;
  result->array              = private_data->parent;  // makes a shallow copy
  result->array.private_data = private_data.release();
  result->array.release      = &detail::ArrowDeviceArrayRelease;
  return result;
}

}  // namespace

unique_device_array_t to_arrow_device(cudf::table&& table,
                                      rmm::cuda_stream_view stream,
                                      rmm::device_async_resource_ref mr)
{
  nanoarrow::UniqueArray tmp;
  NANOARROW_THROW_NOT_OK(ArrowArrayInitFromType(tmp.get(), NANOARROW_TYPE_STRUCT));

  NANOARROW_THROW_NOT_OK(ArrowArrayAllocateChildren(tmp.get(), table.num_columns()));
  tmp->length     = table.num_rows();
  tmp->null_count = 0;

  auto cols = table.release();
  for (size_t i = 0; i < cols.size(); ++i) {
    auto child = tmp->children[i];
    auto col   = cols[i].get();
    NANOARROW_THROW_NOT_OK(cudf::type_dispatcher(
      col->type(), detail::dispatch_to_arrow_device{}, std::move(*col), stream, mr, child));
  }

  return create_device_array(std::move(tmp), stream);
}

unique_device_array_t to_arrow_device(cudf::column&& col,
                                      rmm::cuda_stream_view stream,
                                      rmm::device_async_resource_ref mr)
{
  nanoarrow::UniqueArray tmp;

  NANOARROW_THROW_NOT_OK(cudf::type_dispatcher(
    col.type(), detail::dispatch_to_arrow_device{}, std::move(col), stream, mr, tmp.get()));

  return create_device_array(std::move(tmp), stream);
}

unique_device_array_t to_arrow_device(cudf::table_view const& table,
                                      rmm::cuda_stream_view stream,
                                      rmm::device_async_resource_ref mr)
{
  nanoarrow::UniqueArray tmp;
  NANOARROW_THROW_NOT_OK(ArrowArrayInitFromType(tmp.get(), NANOARROW_TYPE_STRUCT));

  NANOARROW_THROW_NOT_OK(ArrowArrayAllocateChildren(tmp.get(), table.num_columns()));
  tmp->length     = table.num_rows();
  tmp->null_count = 0;

  for (cudf::size_type i = 0; i < table.num_columns(); ++i) {
    auto child = tmp->children[i];
    auto col   = table.column(i);
    NANOARROW_THROW_NOT_OK(cudf::type_dispatcher(
      col.type(), detail::dispatch_to_arrow_device_view{col, stream, mr}, child));
  }

  return create_device_array(std::move(tmp), stream);
}

unique_device_array_t to_arrow_device(cudf::column_view const& col,
                                      rmm::cuda_stream_view stream,
                                      rmm::device_async_resource_ref mr)
{
  nanoarrow::UniqueArray tmp;

  NANOARROW_THROW_NOT_OK(cudf::type_dispatcher(
    col.type(), detail::dispatch_to_arrow_device_view{col, stream, mr}, tmp.get()));

  return create_device_array(std::move(tmp), stream);
}

}  // namespace detail

unique_device_array_t to_arrow_device(cudf::table&& table,
                                      rmm::cuda_stream_view stream,
                                      rmm::device_async_resource_ref mr)
{
  CUDF_FUNC_RANGE();
  return detail::to_arrow_device(std::move(table), stream, mr);
}

unique_device_array_t to_arrow_device(cudf::column&& col,
                                      rmm::cuda_stream_view stream,
                                      rmm::device_async_resource_ref mr)
{
  CUDF_FUNC_RANGE();
  return detail::to_arrow_device(std::move(col), stream, mr);
}

unique_device_array_t to_arrow_device(cudf::table_view const& table,
                                      rmm::cuda_stream_view stream,
                                      rmm::device_async_resource_ref mr)
{
  CUDF_FUNC_RANGE();
  return detail::to_arrow_device(table, stream, mr);
}

unique_device_array_t to_arrow_device(cudf::column_view const& col,
                                      rmm::cuda_stream_view stream,
                                      rmm::device_async_resource_ref mr)
{
  CUDF_FUNC_RANGE();
  return detail::to_arrow_device(col, stream, mr);
}
}  // namespace cudf
