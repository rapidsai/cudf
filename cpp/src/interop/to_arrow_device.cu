#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2024, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <cudf/column/column.hpp>
#include <cudf/column/column_view.hpp>
#include <cudf/detail/interop.hpp>
#include <cudf/detail/iterator.cuh>
#include <cudf/detail/null_mask.hpp>
#include <cudf/dictionary/dictionary_column_view.hpp>
#include <cudf/interop.hpp>
#include <cudf/interop/detail/arrow.hpp>
#include <cudf/lists/lists_column_view.hpp>
#include <cudf/null_mask.hpp>
#include <cudf/strings/strings_column_view.hpp>
#include <cudf/table/table_view.hpp>
#include <cudf/types.hpp>
#include <cudf/utilities/traits.hpp>
#include <cudf/utilities/type_dispatcher.hpp>

#include <rmm/cuda_stream_view.hpp>
#include <rmm/device_scalar.hpp>
#include <rmm/exec_policy.hpp>
#include <rmm/mr/device/per_device_resource.hpp>

#include <thrust/for_each.h>
#include <thrust/iterator/counting_iterator.h>

#include <nanoarrow/nanoarrow.h>
#include <nanoarrow/nanoarrow.hpp>

namespace cudf {
namespace detail {
namespace {
static constexpr int validity_buffer_idx         = 0;
static constexpr int fixed_width_data_buffer_idx = 1;

ArrowType id_to_arrow_type(cudf::type_id id)
{
  switch (id) {
    case cudf::type_id::BOOL8: return NANOARROW_TYPE_BOOL;
    case cudf::type_id::INT8: return NANOARROW_TYPE_INT8;
    case cudf::type_id::INT16: return NANOARROW_TYPE_INT16;
    case cudf::type_id::INT32: return NANOARROW_TYPE_INT32;
    case cudf::type_id::INT64: return NANOARROW_TYPE_INT64;
    case cudf::type_id::UINT8: return NANOARROW_TYPE_UINT8;
    case cudf::type_id::UINT16: return NANOARROW_TYPE_UINT16;
    case cudf::type_id::UINT32: return NANOARROW_TYPE_UINT32;
    case cudf::type_id::UINT64: return NANOARROW_TYPE_UINT64;
    case cudf::type_id::FLOAT32: return NANOARROW_TYPE_FLOAT;
    case cudf::type_id::FLOAT64: return NANOARROW_TYPE_DOUBLE;
    case cudf::type_id::TIMESTAMP_DAYS: return NANOARROW_TYPE_DATE32;
    default: CUDF_FAIL("Unsupported type_id conversion to arrow type");
  }
}

struct dispatch_to_arrow_type {
  template <typename T, CUDF_ENABLE_IF(not is_rep_layout_compatible<T>())>
  int operator()(column_view, column_metadata const&, ArrowSchema*)
  {
    CUDF_FAIL("Unsupported type for to_arrow_schema");
  }

  template <typename T, CUDF_ENABLE_IF(is_rep_layout_compatible<T>())>
  int operator()(column_view input_view, column_metadata const&, ArrowSchema* out)
  {
    cudf::type_id id = input_view.type().id();
    switch (id) {
      case cudf::type_id::TIMESTAMP_SECONDS:
        return ArrowSchemaSetTypeDateTime(
          out, NANOARROW_TYPE_TIMESTAMP, NANOARROW_TIME_UNIT_SECOND, nullptr);
      case cudf::type_id::TIMESTAMP_MILLISECONDS:
        return ArrowSchemaSetTypeDateTime(
          out, NANOARROW_TYPE_TIMESTAMP, NANOARROW_TIME_UNIT_MILLI, nullptr);
      case cudf::type_id::TIMESTAMP_MICROSECONDS:
        return ArrowSchemaSetTypeDateTime(
          out, NANOARROW_TYPE_TIMESTAMP, NANOARROW_TIME_UNIT_MICRO, nullptr);
      case cudf::type_id::TIMESTAMP_NANOSECONDS:
        return ArrowSchemaSetTypeDateTime(
          out, NANOARROW_TYPE_TIMESTAMP, NANOARROW_TIME_UNIT_NANO, nullptr);
      case cudf::type_id::DURATION_SECONDS:
        return ArrowSchemaSetTypeDateTime(
          out, NANOARROW_TYPE_DURATION, NANOARROW_TIME_UNIT_SECOND, nullptr);
      case cudf::type_id::DURATION_MILLISECONDS:
        return ArrowSchemaSetTypeDateTime(
          out, NANOARROW_TYPE_DURATION, NANOARROW_TIME_UNIT_MILLI, nullptr);
      case cudf::type_id::DURATION_MICROSECONDS:
        return ArrowSchemaSetTypeDateTime(
          out, NANOARROW_TYPE_DURATION, NANOARROW_TIME_UNIT_MICRO, nullptr);
      case cudf::type_id::DURATION_NANOSECONDS:
        return ArrowSchemaSetTypeDateTime(
          out, NANOARROW_TYPE_DURATION, NANOARROW_TIME_UNIT_NANO, nullptr);
      default: return ArrowSchemaSetType(out, id_to_arrow_type(id));
    }
  }
};

template <typename DeviceType>
int decimals_to_arrow(column_view input, ArrowSchema* out)
{
  // Arrow doesn't support decimal32/decimal64 currently. decimal128
  // is the smallest that arrow supports besides float32/float64 so we
  // upcast to decimal128.
  return ArrowSchemaSetTypeDecimal(out,
                                   NANOARROW_TYPE_DECIMAL128,
                                   cudf::detail::max_precision<DeviceType>(),
                                   -input.type().scale());
}

template <>
int dispatch_to_arrow_type::operator()<numeric::decimal32>(column_view input,
                                                           column_metadata const&,
                                                           ArrowSchema* out)
{
  using DeviceType = int32_t;
  return decimals_to_arrow<DeviceType>(input, out);
}

template <>
int dispatch_to_arrow_type::operator()<numeric::decimal64>(column_view input,
                                                           column_metadata const&,
                                                           ArrowSchema* out)
{
  using DeviceType = int64_t;
  return decimals_to_arrow<DeviceType>(input, out);
}

template <>
int dispatch_to_arrow_type::operator()<numeric::decimal128>(column_view input,
                                                            column_metadata const&,
                                                            ArrowSchema* out)
{
  using DeviceType = __int128_t;
  return decimals_to_arrow<DeviceType>(input, out);
}

template <>
int dispatch_to_arrow_type::operator()<cudf::string_view>(column_view input,
                                                          column_metadata const&,
                                                          ArrowSchema* out)
{
  return ArrowSchemaSetType(out, NANOARROW_TYPE_STRING);
}

// these forward declarations are needed due to the recursive calls to them
// inside their definitions and in struct_vew for handling children
template <>
int dispatch_to_arrow_type::operator()<cudf::list_view>(column_view input,
                                                        column_metadata const& metadata,
                                                        ArrowSchema* out);

template <>
int dispatch_to_arrow_type::operator()<cudf::dictionary32>(column_view input,
                                                           column_metadata const& metadata,
                                                           ArrowSchema* out);

template <>
int dispatch_to_arrow_type::operator()<cudf::struct_view>(column_view input,
                                                          column_metadata const& metadata,
                                                          ArrowSchema* out)
{
  CUDF_EXPECTS(metadata.children_meta.size() == static_cast<std::size_t>(input.num_children()),
               "Number of field names and number of children doesn't match\n");

  NANOARROW_RETURN_NOT_OK(ArrowSchemaSetTypeStruct(out, input.num_children()));
  for (int i = 0; i < input.num_children(); ++i) {
    auto child = out->children[i];
    auto col   = input.child(i);
    ArrowSchemaInit(child);
    NANOARROW_RETURN_NOT_OK(ArrowSchemaSetName(child, metadata.children_meta[i].name.c_str()));

    child->flags = col.has_nulls() ? ARROW_FLAG_NULLABLE : 0;

    if (col.type().id() == cudf::type_id::EMPTY) {
      NANOARROW_RETURN_NOT_OK(ArrowSchemaSetType(child, NANOARROW_TYPE_NA));
      continue;
    }

    NANOARROW_RETURN_NOT_OK(cudf::type_dispatcher(
      col.type(), detail::dispatch_to_arrow_type{}, col, metadata.children_meta[i], child));
  }

  return NANOARROW_OK;
}

template <>
int dispatch_to_arrow_type::operator()<cudf::list_view>(column_view input,
                                                        column_metadata const& metadata,
                                                        ArrowSchema* out)
{
  NANOARROW_RETURN_NOT_OK(ArrowSchemaSetType(out, NANOARROW_TYPE_LIST));
  auto child = input.child(cudf::lists_column_view::child_column_index);
  ArrowSchemaInit(out->children[0]);
  if (child.type().id() == cudf::type_id::EMPTY) {
    return ArrowSchemaSetType(out->children[0], NANOARROW_TYPE_NA);
  }
  auto child_meta =
    metadata.children_meta.empty() ? column_metadata{"element"} : metadata.children_meta[0];

  out->flags = input.has_nulls() ? ARROW_FLAG_NULLABLE : 0;
  NANOARROW_RETURN_NOT_OK(ArrowSchemaSetName(out->children[0], child_meta.name.c_str()));
  out->children[0]->flags = child.has_nulls() ? ARROW_FLAG_NULLABLE : 0;
  return cudf::type_dispatcher(
    child.type(), detail::dispatch_to_arrow_type{}, child, child_meta, out->children[0]);
}

template <>
int dispatch_to_arrow_type::operator()<cudf::dictionary32>(column_view input,
                                                           column_metadata const& metadata,
                                                           ArrowSchema* out)
{
  cudf::dictionary_column_view dview{input};

  NANOARROW_RETURN_NOT_OK(ArrowSchemaSetType(out, id_to_arrow_type(dview.indices().type().id())));
  NANOARROW_RETURN_NOT_OK(ArrowSchemaAllocateDictionary(out));
  ArrowSchemaInit(out->dictionary);

  auto dict_keys = dview.keys();
  return cudf::type_dispatcher(
    dict_keys.type(),
    detail::dispatch_to_arrow_type{},
    dict_keys,
    metadata.children_meta.empty() ? column_metadata{"keys"} : metadata.children_meta[0],
    out->dictionary);
}

template <typename T>
void device_buffer_finalize(ArrowBufferAllocator* allocator, uint8_t*, int64_t)
{
  auto* unique_buffer = reinterpret_cast<std::unique_ptr<T>*>(allocator->private_data);
  delete unique_buffer;
}

template <typename>
struct is_device_scalar : public std::false_type {};

template <typename T>
struct is_device_scalar<rmm::device_scalar<T>> : public std::true_type {};

template <typename>
struct is_device_uvector : public std::false_type {};

template <typename T>
struct is_device_uvector<rmm::device_uvector<T>> : public std::true_type {};

template <typename T>
int set_buffer(std::unique_ptr<T> device_buf, int64_t i, ArrowArray* out)
{
  ArrowBuffer* buf = ArrowArrayBuffer(out, i);
  auto ptr         = reinterpret_cast<uint8_t*>(device_buf->data());
  buf->size_bytes  = [&] {
    if constexpr (is_device_scalar<T>::value) {
      return sizeof(typename T::value_type);
    } else if constexpr (is_device_uvector<T>::value) {
      return sizeof(typename T::value_type) * device_buf->size();
    } else {
      return device_buf->size();
    }
  }();
  // we make a new unique_ptr and move to it in case there was a custom deleter
  NANOARROW_RETURN_NOT_OK(
    ArrowBufferSetAllocator(buf,
                            ArrowBufferDeallocator(&device_buffer_finalize<T>,
                                                   new std::unique_ptr<T>(std::move(device_buf)))));
  buf->data = ptr;
  return NANOARROW_OK;
}

int initialize_array(ArrowArray* arr, ArrowType storage_type, cudf::column const& column)
{
  NANOARROW_RETURN_NOT_OK(ArrowArrayInitFromType(arr, storage_type));
  arr->length     = column.size();
  arr->null_count = column.null_count();
  return NANOARROW_OK;
}

struct dispatch_to_arrow_device {
  template <typename T, CUDF_ENABLE_IF(not is_rep_layout_compatible<T>())>
  int operator()(cudf::column&&,
                 rmm::cuda_stream_view,
                 rmm::mr::device_memory_resource*,
                 ArrowArray*)
  {
    CUDF_FAIL("Unsupported type for to_arrow_device");
  }

  template <typename T, CUDF_ENABLE_IF(is_rep_layout_compatible<T>())>
  int operator()(cudf::column&& column,
                 rmm::cuda_stream_view stream,
                 rmm::mr::device_memory_resource* mr,
                 ArrowArray* out)
  {
    nanoarrow::UniqueArray tmp;

    const ArrowType storage_type = [&] {
      switch (column.type().id()) {
        case cudf::type_id::TIMESTAMP_SECONDS:
        case cudf::type_id::TIMESTAMP_MILLISECONDS:
        case cudf::type_id::TIMESTAMP_MICROSECONDS:
        case cudf::type_id::TIMESTAMP_NANOSECONDS: return NANOARROW_TYPE_INT64;
        case cudf::type_id::DURATION_SECONDS:
        case cudf::type_id::DURATION_MILLISECONDS:
        case cudf::type_id::DURATION_MICROSECONDS:
        case cudf::type_id::DURATION_NANOSECONDS: return NANOARROW_TYPE_INT64;
        default: return id_to_arrow_type(column.type().id());
      }
    }();
    NANOARROW_RETURN_NOT_OK(initialize_array(tmp.get(), storage_type, column));

    auto contents = column.release();
    if (contents.null_mask) {
      NANOARROW_RETURN_NOT_OK(
        set_buffer(std::move(contents.null_mask), validity_buffer_idx, tmp.get()));
    }

    NANOARROW_RETURN_NOT_OK(
      set_buffer(std::move(contents.data), fixed_width_data_buffer_idx, tmp.get()));

    ArrowArrayMove(tmp.get(), out);
    return NANOARROW_OK;
  }
};

template <typename DeviceType>
int decimals_to_arrow(cudf::column&& input,
                      int32_t precision,
                      rmm::cuda_stream_view stream,
                      rmm::mr::device_memory_resource* mr,
                      ArrowArray* out)
{
  nanoarrow::UniqueArray tmp;
  NANOARROW_RETURN_NOT_OK(initialize_array(tmp.get(), NANOARROW_TYPE_DECIMAL128, input));

  if constexpr (!std::is_same_v<DeviceType, __int128_t>) {
    constexpr size_type BIT_WIDTH_RATIO = sizeof(__int128_t) / sizeof(DeviceType);
    auto buf =
      std::make_unique<rmm::device_uvector<DeviceType>>(input.size() * BIT_WIDTH_RATIO, stream, mr);

    auto count = thrust::make_counting_iterator(0);

    thrust::for_each(rmm::exec_policy(stream, mr),
                     count,
                     count + input.size(),
                     [in  = input.view().begin<DeviceType>(),
                      out = buf->data(),
                      BIT_WIDTH_RATIO] __device__(auto in_idx) {
                       auto const out_idx = in_idx * BIT_WIDTH_RATIO;
                       // the lowest order bits are the value, the remainder
                       // simply matches the sign bit to satisfy the two's
                       // complement integer representation of negative numbers.
                       out[out_idx] = in[in_idx];
#pragma unroll BIT_WIDTH_RATIO - 1
                       for (auto i = 1; i < BIT_WIDTH_RATIO; ++i) {
                         out[out_idx + i] = in[in_idx] < 0 ? -1 : 0;
                       }
                     });
    NANOARROW_RETURN_NOT_OK(set_buffer(std::move(buf), fixed_width_data_buffer_idx, tmp.get()));
  }

  auto contents = input.release();
  if (contents.null_mask) {
    NANOARROW_RETURN_NOT_OK(
      set_buffer(std::move(contents.null_mask), validity_buffer_idx, tmp.get()));
  }

  if constexpr (std::is_same_v<DeviceType, __int128_t>) {
    NANOARROW_RETURN_NOT_OK(
      set_buffer(std::move(contents.data), fixed_width_data_buffer_idx, tmp.get()));
  }

  ArrowArrayMove(tmp.get(), out);
  return NANOARROW_OK;
}

template <>
int dispatch_to_arrow_device::operator()<numeric::decimal32>(cudf::column&& column,
                                                             rmm::cuda_stream_view stream,
                                                             rmm::mr::device_memory_resource* mr,
                                                             ArrowArray* out)
{
  using DeviceType = int32_t;
  return decimals_to_arrow<DeviceType>(
    std::move(column), cudf::detail::max_precision<DeviceType>(), stream, mr, out);
}

template <>
int dispatch_to_arrow_device::operator()<numeric::decimal64>(cudf::column&& column,
                                                             rmm::cuda_stream_view stream,
                                                             rmm::mr::device_memory_resource* mr,
                                                             ArrowArray* out)
{
  using DeviceType = int64_t;
  return decimals_to_arrow<DeviceType>(
    std::move(column), cudf::detail::max_precision<DeviceType>(), stream, mr, out);
}

template <>
int dispatch_to_arrow_device::operator()<numeric::decimal128>(cudf::column&& column,
                                                              rmm::cuda_stream_view stream,
                                                              rmm::mr::device_memory_resource* mr,
                                                              ArrowArray* out)
{
  using DeviceType = __int128_t;
  return decimals_to_arrow<DeviceType>(
    std::move(column), cudf::detail::max_precision<DeviceType>(), stream, mr, out);
}

template <>
int dispatch_to_arrow_device::operator()<bool>(cudf::column&& column,
                                               rmm::cuda_stream_view stream,
                                               rmm::mr::device_memory_resource* mr,
                                               ArrowArray* out)
{
  nanoarrow::UniqueArray tmp;
  NANOARROW_RETURN_NOT_OK(initialize_array(tmp.get(), NANOARROW_TYPE_BOOL, column));

  auto bitmask  = bools_to_mask(column.view(), stream, mr);
  auto contents = column.release();
  if (contents.null_mask) {
    NANOARROW_RETURN_NOT_OK(
      set_buffer(std::move(contents.null_mask), validity_buffer_idx, tmp.get()));
  }
  NANOARROW_RETURN_NOT_OK(
    set_buffer(std::move(bitmask.first), fixed_width_data_buffer_idx, tmp.get()));

  ArrowArrayMove(tmp.get(), out);
  return NANOARROW_OK;
}

template <>
int dispatch_to_arrow_device::operator()<cudf::string_view>(cudf::column&& column,
                                                            rmm::cuda_stream_view stream,
                                                            rmm::mr::device_memory_resource* mr,
                                                            ArrowArray* out)
{
  nanoarrow::UniqueArray tmp;
  NANOARROW_RETURN_NOT_OK(initialize_array(tmp.get(), NANOARROW_TYPE_STRING, column));

  if (column.size() == 0) {
    // the scalar zero here is necessary because the spec for string arrays states
    // that the offsets buffer should contain "length + 1" signed integers. So in
    // the case of a 0 length string array, there should be exactly 1 value, zero,
    // in the offsets buffer. While some arrow implementations may accept a zero-sized
    // offsets buffer, best practices would be to allocate the buffer with the single value.
    auto zero = std::make_unique<rmm::device_scalar<int32_t>>(0, stream, mr);
    NANOARROW_RETURN_NOT_OK(set_buffer(std::move(zero), fixed_width_data_buffer_idx, tmp.get()));
    ArrowArrayMove(tmp.get(), out);
    return NANOARROW_OK;
  }

  auto contents = column.release();
  if (contents.null_mask) {
    NANOARROW_RETURN_NOT_OK(
      set_buffer(std::move(contents.null_mask), validity_buffer_idx, tmp.get()));
  }

  auto offsets_contents =
    contents.children[cudf::strings_column_view::offsets_column_index]->release();
  NANOARROW_RETURN_NOT_OK(set_buffer(std::move(offsets_contents.data), 1, tmp.get()));
  NANOARROW_RETURN_NOT_OK(set_buffer(std::move(contents.data), 2, tmp.get()));

  ArrowArrayMove(tmp.get(), out);
  return NANOARROW_OK;
}

template <>
int dispatch_to_arrow_device::operator()<cudf::list_view>(cudf::column&& column,
                                                          rmm::cuda_stream_view stream,
                                                          rmm::mr::device_memory_resource* mr,
                                                          ArrowArray* out);

template <>
int dispatch_to_arrow_device::operator()<cudf::dictionary32>(cudf::column&& column,
                                                             rmm::cuda_stream_view stream,
                                                             rmm::mr::device_memory_resource* mr,
                                                             ArrowArray* out);

template <>
int dispatch_to_arrow_device::operator()<cudf::struct_view>(cudf::column&& column,
                                                            rmm::cuda_stream_view stream,
                                                            rmm::mr::device_memory_resource* mr,
                                                            ArrowArray* out)
{
  nanoarrow::UniqueArray tmp;
  NANOARROW_RETURN_NOT_OK(initialize_array(tmp.get(), NANOARROW_TYPE_STRUCT, column));
  NANOARROW_RETURN_NOT_OK(ArrowArrayAllocateChildren(tmp.get(), column.num_children()));

  auto contents = column.release();
  if (contents.null_mask) {
    NANOARROW_RETURN_NOT_OK(
      set_buffer(std::move(contents.null_mask), validity_buffer_idx, tmp.get()));
  }

  for (size_t i = 0; i < size_t(tmp->n_children); ++i) {
    ArrowArray* child_ptr = tmp->children[i];
    auto& child           = contents.children[i];
    if (child->type().id() == cudf::type_id::EMPTY) {
      NANOARROW_RETURN_NOT_OK(ArrowArrayInitFromType(child_ptr, NANOARROW_TYPE_NA));
      child_ptr->length     = child->size();
      child_ptr->null_count = child->size();
    } else {
      NANOARROW_RETURN_NOT_OK(cudf::type_dispatcher(
        child->type(), dispatch_to_arrow_device{}, std::move(*child), stream, mr, child_ptr));
    }
  }

  ArrowArrayMove(tmp.get(), out);
  return NANOARROW_OK;
}

template <>
int dispatch_to_arrow_device::operator()<cudf::list_view>(cudf::column&& column,
                                                          rmm::cuda_stream_view stream,
                                                          rmm::mr::device_memory_resource* mr,
                                                          ArrowArray* out)
{
  nanoarrow::UniqueArray tmp;
  NANOARROW_RETURN_NOT_OK(initialize_array(tmp.get(), NANOARROW_TYPE_LIST, column));
  NANOARROW_RETURN_NOT_OK(ArrowArrayAllocateChildren(tmp.get(), 1));

  auto contents = column.release();
  if (contents.null_mask) {
    NANOARROW_RETURN_NOT_OK(
      set_buffer(std::move(contents.null_mask), validity_buffer_idx, tmp.get()));
  }

  auto offsets_contents =
    contents.children[cudf::lists_column_view::offsets_column_index]->release();
  NANOARROW_RETURN_NOT_OK(set_buffer(std::move(offsets_contents.data), 1, tmp.get()));

  auto& child = contents.children[cudf::lists_column_view::child_column_index];
  if (child->type().id() == cudf::type_id::EMPTY) {
    NANOARROW_RETURN_NOT_OK(ArrowArrayInitFromType(tmp->children[0], NANOARROW_TYPE_NA));
    tmp->children[0]->length     = 0;
    tmp->children[0]->null_count = 0;
  } else {
    NANOARROW_RETURN_NOT_OK(cudf::type_dispatcher(
      child->type(), dispatch_to_arrow_device{}, std::move(*child), stream, mr, tmp->children[0]));
  }

  ArrowArrayMove(tmp.get(), out);
  return NANOARROW_OK;
}

template <>
int dispatch_to_arrow_device::operator()<cudf::dictionary32>(cudf::column&& column,
                                                             rmm::cuda_stream_view stream,
                                                             rmm::mr::device_memory_resource* mr,
                                                             ArrowArray* out)
{
  nanoarrow::UniqueArray tmp;
  NANOARROW_RETURN_NOT_OK(initialize_array(
    tmp.get(),
    id_to_arrow_type(column.child(cudf::dictionary_column_view::indices_column_index).type().id()),
    column));
  NANOARROW_RETURN_NOT_OK(ArrowArrayAllocateDictionary(tmp.get()));

  auto contents = column.release();
  if (contents.null_mask) {
    NANOARROW_RETURN_NOT_OK(
      set_buffer(std::move(contents.null_mask), validity_buffer_idx, tmp.get()));
  }

  auto indices_contents =
    contents.children[cudf::dictionary_column_view::indices_column_index]->release();
  NANOARROW_RETURN_NOT_OK(
    set_buffer(std::move(indices_contents.data), fixed_width_data_buffer_idx, tmp.get()));

  auto& keys = contents.children[cudf::dictionary_column_view::keys_column_index];
  NANOARROW_RETURN_NOT_OK(cudf::type_dispatcher(
    keys->type(), dispatch_to_arrow_device{}, std::move(*keys), stream, mr, tmp->dictionary));

  ArrowArrayMove(tmp.get(), out);
  return NANOARROW_OK;
}

struct ArrowDeviceArrayPrivateData {
  ArrowArray parent;
  hipEvent_t sync_event;
};

void ArrowDeviceArrayRelease(ArrowArray* array)
{
  auto private_data = reinterpret_cast<ArrowDeviceArrayPrivateData*>(array->private_data);
  hipEventDestroy(private_data->sync_event);
  ArrowArrayRelease(&private_data->parent);
  delete private_data;
  array->release = nullptr;
}

}  // namespace
}  // namespace detail

unique_schema_t to_arrow_schema(cudf::table_view const& input,
                                cudf::host_span<column_metadata const> metadata)
{
  CUDF_EXPECTS((metadata.size() == static_cast<std::size_t>(input.num_columns())),
               "columns' metadata should be equal to the number of columns in table");

  nanoarrow::UniqueSchema result;
  ArrowSchemaInit(result.get());
  NANOARROW_THROW_NOT_OK(ArrowSchemaSetTypeStruct(result.get(), input.num_columns()));

  for (int i = 0; i < input.num_columns(); ++i) {
    auto child = result->children[i];
    auto col   = input.column(i);
    ArrowSchemaInit(child);
    NANOARROW_THROW_NOT_OK(ArrowSchemaSetName(child, metadata[i].name.c_str()));
    child->flags = col.has_nulls() ? ARROW_FLAG_NULLABLE : 0;

    if (col.type().id() == cudf::type_id::EMPTY) {
      NANOARROW_THROW_NOT_OK(ArrowSchemaSetType(child, NANOARROW_TYPE_NA));
      continue;
    }

    NANOARROW_THROW_NOT_OK(
      cudf::type_dispatcher(col.type(), detail::dispatch_to_arrow_type{}, col, metadata[i], child));
  }

  unique_schema_t out(new ArrowSchema, [](ArrowSchema* schema) {
    if (schema->release != nullptr) { ArrowSchemaRelease(schema); }
    delete schema;
  });
  result.move(out.get());
  return out;
}

unique_device_array_t to_arrow_device(cudf::table&& table,
                                      rmm::cuda_stream_view stream,
                                      rmm::mr::device_memory_resource* mr)
{
  nanoarrow::UniqueArray tmp;
  NANOARROW_THROW_NOT_OK(ArrowArrayInitFromType(tmp.get(), NANOARROW_TYPE_STRUCT));

  NANOARROW_THROW_NOT_OK(ArrowArrayAllocateChildren(tmp.get(), table.num_columns()));
  tmp->length     = table.num_rows();
  tmp->null_count = 0;

  auto cols = table.release();
  for (size_t i = 0; i < cols.size(); ++i) {
    auto child = tmp->children[i];
    auto col   = cols[i].get();

    if (col->type().id() == cudf::type_id::EMPTY) {
      NANOARROW_THROW_NOT_OK(ArrowArrayInitFromType(child, NANOARROW_TYPE_NA));
      child->length     = col->size();
      child->null_count = col->size();
      continue;
    }

    NANOARROW_THROW_NOT_OK(cudf::type_dispatcher(
      col->type(), detail::dispatch_to_arrow_device{}, std::move(*col), stream, mr, child));
  }

  NANOARROW_THROW_NOT_OK(
    ArrowArrayFinishBuilding(tmp.get(), NANOARROW_VALIDATION_LEVEL_MINIMAL, nullptr));

  auto private_data = std::make_unique<detail::ArrowDeviceArrayPrivateData>();
  hipEventCreate(&private_data->sync_event);

  auto status = hipEventRecord(private_data->sync_event, stream);
  if (status != hipSuccess) { CUDF_FAIL("could not create event to sync on"); }

  ArrowArrayMove(tmp.get(), &private_data->parent);
  unique_device_array_t result(new ArrowDeviceArray, [](ArrowDeviceArray* arr) {
    if (arr->array.release != nullptr) { ArrowArrayRelease(&arr->array); }
    delete arr;
  });
  result->device_id          = rmm::get_current_cuda_device().value();
  result->device_type        = ARROW_DEVICE_CUDA;
  result->sync_event         = &private_data->sync_event;
  result->array              = private_data->parent;
  result->array.private_data = private_data.release();
  result->array.release      = &detail::ArrowDeviceArrayRelease;
  return result;
}

unique_device_array_t to_arrow_device(cudf::column&& col,
                                      rmm::cuda_stream_view stream,
                                      rmm::mr::device_memory_resource* mr)
{
  nanoarrow::UniqueArray tmp;
  if (col.type().id() == cudf::type_id::EMPTY) {
    NANOARROW_THROW_NOT_OK(ArrowArrayInitFromType(tmp.get(), NANOARROW_TYPE_NA));
    tmp->length     = col.size();
    tmp->null_count = col.size();
  }

  NANOARROW_THROW_NOT_OK(cudf::type_dispatcher(
    col.type(), detail::dispatch_to_arrow_device{}, std::move(col), stream, mr, tmp.get()));

  NANOARROW_THROW_NOT_OK(
    ArrowArrayFinishBuilding(tmp.get(), NANOARROW_VALIDATION_LEVEL_MINIMAL, nullptr));

  auto private_data = std::make_unique<detail::ArrowDeviceArrayPrivateData>();
  hipEventCreate(&private_data->sync_event);

  auto status = hipEventRecord(private_data->sync_event, stream);
  if (status != hipSuccess) { CUDF_FAIL("could not create event to sync on"); }

  ArrowArrayMove(tmp.get(), &private_data->parent);
  unique_device_array_t result(new ArrowDeviceArray, [](ArrowDeviceArray* arr) {
    if (arr->array.release != nullptr) { ArrowArrayRelease(&arr->array); }
    delete arr;
  });
  result->device_id          = rmm::get_current_cuda_device().value();
  result->device_type        = ARROW_DEVICE_CUDA;
  result->sync_event         = &private_data->sync_event;
  result->array              = private_data->parent;
  result->array.private_data = private_data.release();
  result->array.release      = &detail::ArrowDeviceArrayRelease;
  return result;
}

}  // namespace cudf
