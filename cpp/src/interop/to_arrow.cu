/*
 * Copyright (c) 2020-2024, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "arrow_utilities.hpp"
#include "decimal_conversion_utilities.cuh"
#include "detail/arrow_allocator.hpp"

#include <cudf/column/column.hpp>
#include <cudf/column/column_factories.hpp>
#include <cudf/column/column_view.hpp>
#include <cudf/copying.hpp>
#include <cudf/detail/interop.hpp>
#include <cudf/detail/iterator.cuh>
#include <cudf/detail/null_mask.hpp>
#include <cudf/detail/nvtx/ranges.hpp>
#include <cudf/detail/unary.hpp>
#include <cudf/dictionary/dictionary_column_view.hpp>
#include <cudf/interop.hpp>
#include <cudf/null_mask.hpp>
#include <cudf/strings/strings_column_view.hpp>
#include <cudf/table/table_view.hpp>
#include <cudf/types.hpp>
#include <cudf/utilities/default_stream.hpp>
#include <cudf/utilities/traits.hpp>
#include <cudf/utilities/type_dispatcher.hpp>

#include <rmm/cuda_stream_view.hpp>
#include <rmm/exec_policy.hpp>
#include <rmm/mr/device/per_device_resource.hpp>

#include <thrust/copy.h>
#include <thrust/for_each.h>
#include <thrust/iterator/counting_iterator.h>

namespace cudf {
namespace detail {
namespace {

/**
 * @brief Create arrow data buffer from given cudf column
 */
template <typename T>
std::shared_ptr<arrow::Buffer> fetch_data_buffer(device_span<T const> input,
                                                 arrow::MemoryPool* ar_mr,
                                                 rmm::cuda_stream_view stream)
{
  int64_t const data_size_in_bytes = sizeof(T) * input.size();

  auto data_buffer = allocate_arrow_buffer(data_size_in_bytes, ar_mr);

  CUDF_CUDA_TRY(hipMemcpyAsync(data_buffer->mutable_data(),
                                input.data(),
                                data_size_in_bytes,
                                hipMemcpyDefault,
                                stream.value()));

  return std::move(data_buffer);
}

/**
 * @brief Create arrow buffer of mask from given cudf column
 */
std::shared_ptr<arrow::Buffer> fetch_mask_buffer(column_view input_view,
                                                 arrow::MemoryPool* ar_mr,
                                                 rmm::cuda_stream_view stream)
{
  int64_t const mask_size_in_bytes = cudf::bitmask_allocation_size_bytes(input_view.size());

  if (input_view.has_nulls()) {
    auto mask_buffer = allocate_arrow_bitmap(static_cast<int64_t>(input_view.size()), ar_mr);
    CUDF_CUDA_TRY(hipMemcpyAsync(
      mask_buffer->mutable_data(),
      (input_view.offset() > 0)
        ? cudf::detail::copy_bitmask(input_view, stream, rmm::mr::get_current_device_resource())
            .data()
        : input_view.null_mask(),
      mask_size_in_bytes,
      hipMemcpyDefault,
      stream.value()));

    // Resets all padded bits to 0
    mask_buffer->ZeroPadding();

    return mask_buffer;
  }

  return nullptr;
}

/**
 * @brief Functor to convert cudf column to arrow array
 */
struct dispatch_to_arrow {
  /**
   * @brief Creates vector Arrays from given cudf column children
   */
  std::vector<std::shared_ptr<arrow::Array>> fetch_child_array(
    column_view input_view,
    std::vector<column_metadata> const& metadata,
    arrow::MemoryPool* ar_mr,
    rmm::cuda_stream_view stream)
  {
    std::vector<std::shared_ptr<arrow::Array>> child_arrays;
    std::transform(
      input_view.child_begin(),
      input_view.child_end(),
      metadata.begin(),
      std::back_inserter(child_arrays),
      [&ar_mr, &stream](auto const& child, auto const& meta) {
        return type_dispatcher(
          child.type(), dispatch_to_arrow{}, child, child.type().id(), meta, ar_mr, stream);
      });
    return child_arrays;
  }

  template <typename T, CUDF_ENABLE_IF(not is_rep_layout_compatible<T>())>
  std::shared_ptr<arrow::Array> operator()(
    column_view, cudf::type_id, column_metadata const&, arrow::MemoryPool*, rmm::cuda_stream_view)
  {
    CUDF_FAIL("Unsupported type for to_arrow.");
  }

  template <typename T, CUDF_ENABLE_IF(is_rep_layout_compatible<T>())>
  std::shared_ptr<arrow::Array> operator()(column_view input_view,
                                           cudf::type_id id,
                                           column_metadata const&,
                                           arrow::MemoryPool* ar_mr,
                                           rmm::cuda_stream_view stream)
  {
    return to_arrow_array(
      id,
      static_cast<int64_t>(input_view.size()),
      fetch_data_buffer<T>(
        device_span<T const>(input_view.data<T>(), input_view.size()), ar_mr, stream),
      fetch_mask_buffer(input_view, ar_mr, stream),
      static_cast<int64_t>(input_view.null_count()));
  }
};

// Convert decimal types from libcudf to arrow where those types are not
// directly supported by Arrow. These types must be fit into 128 bits, the
// smallest decimal resolution supported by Arrow.
template <typename DeviceType>
std::shared_ptr<arrow::Array> unsupported_decimals_to_arrow(column_view input,
                                                            int32_t precision,
                                                            arrow::MemoryPool* ar_mr,
                                                            rmm::cuda_stream_view stream)
{
  auto buf = detail::convert_decimals_to_decimal128<DeviceType>(
    input, stream, rmm::mr::get_current_device_resource());

  // Synchronize stream here to ensure the decimal128 buffer is ready.
  stream.synchronize();

  auto const buf_size_in_bytes = buf->size();
  auto data_buffer             = allocate_arrow_buffer(buf_size_in_bytes, ar_mr);

  CUDF_CUDA_TRY(hipMemcpyAsync(data_buffer->mutable_data(),
                                buf->data(),
                                buf_size_in_bytes,
                                hipMemcpyDefault,
                                stream.value()));

  auto type    = arrow::decimal(precision, -input.type().scale());
  auto mask    = fetch_mask_buffer(input, ar_mr, stream);
  auto buffers = std::vector<std::shared_ptr<arrow::Buffer>>{mask, std::move(data_buffer)};
  auto data    = std::make_shared<arrow::ArrayData>(type, input.size(), buffers);

  return std::make_shared<arrow::Decimal128Array>(data);
}

template <>
std::shared_ptr<arrow::Array> dispatch_to_arrow::operator()<numeric::decimal32>(
  column_view input,
  cudf::type_id,
  column_metadata const&,
  arrow::MemoryPool* ar_mr,
  rmm::cuda_stream_view stream)
{
  using DeviceType = int32_t;
  return unsupported_decimals_to_arrow<DeviceType>(
    input, cudf::detail::max_precision<DeviceType>(), ar_mr, stream);
}

template <>
std::shared_ptr<arrow::Array> dispatch_to_arrow::operator()<numeric::decimal64>(
  column_view input,
  cudf::type_id,
  column_metadata const&,
  arrow::MemoryPool* ar_mr,
  rmm::cuda_stream_view stream)
{
  using DeviceType = int64_t;
  return unsupported_decimals_to_arrow<DeviceType>(
    input, cudf::detail::max_precision<DeviceType>(), ar_mr, stream);
}

template <>
std::shared_ptr<arrow::Array> dispatch_to_arrow::operator()<numeric::decimal128>(
  column_view input,
  cudf::type_id,
  column_metadata const&,
  arrow::MemoryPool* ar_mr,
  rmm::cuda_stream_view stream)
{
  using DeviceType         = __int128_t;
  auto const max_precision = cudf::detail::max_precision<DeviceType>();

  rmm::device_uvector<DeviceType> buf(input.size(), stream);

  thrust::copy(rmm::exec_policy(stream),  //
               input.begin<DeviceType>(),
               input.end<DeviceType>(),
               buf.begin());

  auto const buf_size_in_bytes = buf.size() * sizeof(DeviceType);
  auto data_buffer             = allocate_arrow_buffer(buf_size_in_bytes, ar_mr);

  CUDF_CUDA_TRY(hipMemcpyAsync(
    data_buffer->mutable_data(), buf.data(), buf_size_in_bytes, hipMemcpyDefault, stream.value()));

  auto type    = arrow::decimal(max_precision, -input.type().scale());
  auto mask    = fetch_mask_buffer(input, ar_mr, stream);
  auto buffers = std::vector<std::shared_ptr<arrow::Buffer>>{mask, std::move(data_buffer)};
  auto data    = std::make_shared<arrow::ArrayData>(type, input.size(), buffers);

  return std::make_shared<arrow::Decimal128Array>(data);
}

template <>
std::shared_ptr<arrow::Array> dispatch_to_arrow::operator()<bool>(column_view input,
                                                                  cudf::type_id id,
                                                                  column_metadata const&,
                                                                  arrow::MemoryPool* ar_mr,
                                                                  rmm::cuda_stream_view stream)
{
  auto bitmask = detail::bools_to_mask(input, stream, rmm::mr::get_current_device_resource());

  auto data_buffer = allocate_arrow_buffer(static_cast<int64_t>(bitmask.first->size()), ar_mr);

  CUDF_CUDA_TRY(hipMemcpyAsync(data_buffer->mutable_data(),
                                bitmask.first->data(),
                                bitmask.first->size(),
                                hipMemcpyDefault,
                                stream.value()));
  return to_arrow_array(id,
                        static_cast<int64_t>(input.size()),
                        std::move(data_buffer),
                        fetch_mask_buffer(input, ar_mr, stream),
                        static_cast<int64_t>(input.null_count()));
}

template <>
std::shared_ptr<arrow::Array> dispatch_to_arrow::operator()<cudf::string_view>(
  column_view input,
  cudf::type_id,
  column_metadata const&,
  arrow::MemoryPool* ar_mr,
  rmm::cuda_stream_view stream)
{
  std::unique_ptr<column> tmp_column =
    ((input.offset() != 0) or
     ((input.num_children() == 1) and (input.child(0).size() - 1 != input.size())))
      ? std::make_unique<cudf::column>(input, stream)
      : nullptr;

  column_view input_view = (tmp_column != nullptr) ? tmp_column->view() : input;
  auto child_arrays      = fetch_child_array(input_view, {{}, {}}, ar_mr, stream);
  if (child_arrays.empty()) {
    // Empty string will have only one value in offset of 4 bytes
    auto tmp_offset_buffer = allocate_arrow_buffer(sizeof(int32_t), ar_mr);
    auto tmp_data_buffer   = allocate_arrow_buffer(0, ar_mr);
    memset(tmp_offset_buffer->mutable_data(), 0, sizeof(int32_t));

    return std::make_shared<arrow::StringArray>(
      0, std::move(tmp_offset_buffer), std::move(tmp_data_buffer));
  }
  auto offset_buffer = child_arrays[strings_column_view::offsets_column_index]->data()->buffers[1];
  auto const sview   = strings_column_view{input_view};
  auto data_buffer   = fetch_data_buffer<char>(
    device_span<char const>{sview.chars_begin(stream),
                              static_cast<std::size_t>(sview.chars_size(stream))},
    ar_mr,
    stream);
  if (sview.offsets().type().id() == cudf::type_id::INT64) {
    return std::make_shared<arrow::LargeStringArray>(static_cast<int64_t>(input_view.size()),
                                                     offset_buffer,
                                                     data_buffer,
                                                     fetch_mask_buffer(input_view, ar_mr, stream),
                                                     static_cast<int64_t>(input_view.null_count()));
  } else {
    return std::make_shared<arrow::StringArray>(static_cast<int64_t>(input_view.size()),
                                                offset_buffer,
                                                data_buffer,
                                                fetch_mask_buffer(input_view, ar_mr, stream),
                                                static_cast<int64_t>(input_view.null_count()));
  }
}

template <>
std::shared_ptr<arrow::Array> dispatch_to_arrow::operator()<cudf::struct_view>(
  column_view input,
  cudf::type_id,
  column_metadata const& metadata,
  arrow::MemoryPool* ar_mr,
  rmm::cuda_stream_view stream)
{
  CUDF_EXPECTS(metadata.children_meta.size() == static_cast<std::size_t>(input.num_children()),
               "Number of field names and number of children doesn't match\n");
  std::unique_ptr<column> tmp_column = nullptr;

  if (input.offset() != 0) { tmp_column = std::make_unique<cudf::column>(input, stream); }

  column_view input_view = (tmp_column != nullptr) ? tmp_column->view() : input;
  auto child_arrays      = fetch_child_array(input_view, metadata.children_meta, ar_mr, stream);
  auto mask              = fetch_mask_buffer(input_view, ar_mr, stream);

  std::vector<std::shared_ptr<arrow::Field>> fields;
  std::transform(child_arrays.cbegin(),
                 child_arrays.cend(),
                 metadata.children_meta.cbegin(),
                 std::back_inserter(fields),
                 [](auto const array, auto const meta) {
                   return std::make_shared<arrow::Field>(
                     meta.name, array->type(), array->null_count() > 0);
                 });
  auto dtype = std::make_shared<arrow::StructType>(fields);

  return std::make_shared<arrow::StructArray>(dtype,
                                              static_cast<int64_t>(input_view.size()),
                                              child_arrays,
                                              mask,
                                              static_cast<int64_t>(input_view.null_count()));
}

template <>
std::shared_ptr<arrow::Array> dispatch_to_arrow::operator()<cudf::list_view>(
  column_view input,
  cudf::type_id,
  column_metadata const& metadata,
  arrow::MemoryPool* ar_mr,
  rmm::cuda_stream_view stream)
{
  CUDF_EXPECTS(metadata.children_meta.empty() ||
                 metadata.children_meta.size() == static_cast<std::size_t>(input.num_children()),
               "Number of field names and number of children do not match\n");
  std::unique_ptr<column> tmp_column = nullptr;
  if ((input.offset() != 0) or
      ((input.num_children() == 2) and (input.child(0).size() - 1 != input.size()))) {
    tmp_column = std::make_unique<cudf::column>(input, stream);
  }

  column_view input_view = (tmp_column != nullptr) ? tmp_column->view() : input;
  auto children_meta =
    metadata.children_meta.empty() ? std::vector<column_metadata>{{}, {}} : metadata.children_meta;
  auto child_arrays = fetch_child_array(input_view, children_meta, ar_mr, stream);
  if (child_arrays.empty() || child_arrays[0]->data()->length == 0) {
    auto element_type = child_arrays.empty() ? arrow::null() : child_arrays[1]->type();
    auto result       = arrow::MakeEmptyArray(arrow::list(element_type), ar_mr);
    CUDF_EXPECTS(result.ok(), "Failed to construct empty arrow list array\n");
    return result.ValueUnsafe();
  }

  auto offset_buffer = child_arrays[0]->data()->buffers[1];
  auto data          = child_arrays[1];
  return std::make_shared<arrow::ListArray>(arrow::list(data->type()),
                                            static_cast<int64_t>(input_view.size()),
                                            offset_buffer,
                                            data,
                                            fetch_mask_buffer(input_view, ar_mr, stream),
                                            static_cast<int64_t>(input_view.null_count()));
}

template <>
std::shared_ptr<arrow::Array> dispatch_to_arrow::operator()<cudf::dictionary32>(
  column_view input,
  cudf::type_id,
  column_metadata const& metadata,
  arrow::MemoryPool* ar_mr,
  rmm::cuda_stream_view stream)
{
  // Arrow dictionary requires indices to be signed integer
  std::unique_ptr<column> dict_indices =
    detail::cast(cudf::dictionary_column_view(input).get_indices_annotated(),
                 cudf::data_type{type_id::INT32},
                 stream,
                 rmm::mr::get_current_device_resource());
  auto indices = dispatch_to_arrow{}.operator()<int32_t>(
    dict_indices->view(), dict_indices->type().id(), {}, ar_mr, stream);
  auto dict_keys = cudf::dictionary_column_view(input).keys();
  auto dictionary =
    type_dispatcher(dict_keys.type(),
                    dispatch_to_arrow{},
                    dict_keys,
                    dict_keys.type().id(),
                    metadata.children_meta.empty() ? column_metadata{} : metadata.children_meta[0],
                    ar_mr,
                    stream);

  return std::make_shared<arrow::DictionaryArray>(
    arrow::dictionary(indices->type(), dictionary->type()), indices, dictionary);
}
}  // namespace

std::shared_ptr<arrow::Table> to_arrow(table_view input,
                                       std::vector<column_metadata> const& metadata,
                                       rmm::cuda_stream_view stream,
                                       arrow::MemoryPool* ar_mr)
{
  CUDF_EXPECTS((metadata.size() == static_cast<std::size_t>(input.num_columns())),
               "columns' metadata should be equal to number of columns in table");

  std::vector<std::shared_ptr<arrow::Array>> arrays;
  std::vector<std::shared_ptr<arrow::Field>> fields;

  std::transform(
    input.begin(),
    input.end(),
    metadata.begin(),
    std::back_inserter(arrays),
    [&](auto const& c, auto const& meta) {
      return c.type().id() != type_id::EMPTY
               ? type_dispatcher(
                   c.type(), detail::dispatch_to_arrow{}, c, c.type().id(), meta, ar_mr, stream)
               : std::make_shared<arrow::NullArray>(c.size());
    });

  std::transform(
    arrays.begin(),
    arrays.end(),
    metadata.begin(),
    std::back_inserter(fields),
    [](auto const& array, auto const& meta) { return arrow::field(meta.name, array->type()); });

  auto result = arrow::Table::Make(arrow::schema(fields), arrays);

  // synchronize the stream because after the return the data may be accessed from the host before
  // the above `hipMemcpyAsync` calls have completed their copies (especially if pinned host
  // memory is used).
  stream.synchronize();

  return result;
}

std::shared_ptr<arrow::Scalar> to_arrow(cudf::scalar const& input,
                                        column_metadata const& metadata,
                                        rmm::cuda_stream_view stream,
                                        arrow::MemoryPool* ar_mr)
{
  auto const column = cudf::make_column_from_scalar(input, 1, stream);
  cudf::table_view const tv{{column->view()}};
  auto const arrow_table  = detail::to_arrow(tv, {metadata}, stream, ar_mr);
  auto const ac           = arrow_table->column(0);
  auto const maybe_scalar = ac->GetScalar(0);
  if (!maybe_scalar.ok()) { CUDF_FAIL("Failed to produce a scalar"); }
  return maybe_scalar.ValueOrDie();
}
}  // namespace detail

std::shared_ptr<arrow::Table> to_arrow(table_view input,
                                       std::vector<column_metadata> const& metadata,
                                       rmm::cuda_stream_view stream,
                                       arrow::MemoryPool* ar_mr)
{
  CUDF_FUNC_RANGE();
  return detail::to_arrow(input, metadata, stream, ar_mr);
}

std::shared_ptr<arrow::Scalar> to_arrow(cudf::scalar const& input,
                                        column_metadata const& metadata,
                                        rmm::cuda_stream_view stream,
                                        arrow::MemoryPool* ar_mr)
{
  CUDF_FUNC_RANGE();
  return detail::to_arrow(input, metadata, stream, ar_mr);
}
}  // namespace cudf
