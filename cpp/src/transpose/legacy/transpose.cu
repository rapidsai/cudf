#include "hip/hip_runtime.h"
/* Copyright 2018 NVIDIA Corporation.  All rights reserved. */

#include <cudf/utilities/nvtx_utils.hpp>
#include <cudf/utilities/legacy/type_dispatcher.hpp>
#include <rmm/thrust_rmm_allocator.h>
#include <bitmask/legacy/legacy_bitmask.hpp>
#include <cudf/cudf.h>
#include <hipcub/hipcub.hpp>
#include <memory>
#include <stdio.h>
#include <algorithm>

namespace
{

constexpr int WARP_SIZE = 32;
constexpr int MAX_GRID_SIZE = (1<<16)-1;

/**
 * @brief Transposes the values from ncols x nrows input columns to
 *  nrows x ncols output columns
 * 
 * @tparam ColumnType  Datatype of values pointed to by the pointers
 * @param in_cols[in]  Pointers to input columns' data
 * @param out_cols[out]  Pointers to pre-allocated output columns' data
 * @param ncols[in]  Number of columns in input table
 * @param nrows[in]  Number of rown in input table
 */
template <typename ColumnType>
__global__
void gpu_transpose(ColumnType **in_cols, ColumnType **out_cols,
                  cudf::size_type ncols, cudf::size_type nrows)
{
  cudf::size_type x = blockIdx.x * blockDim.x + threadIdx.x;
  cudf::size_type y = blockIdx.y * blockDim.y + threadIdx.y;
    
  cudf::size_type stride_x = blockDim.x * gridDim.x;
  cudf::size_type stride_y = blockDim.y * gridDim.y;

  for(cudf::size_type i = x; i < ncols; i += stride_x)
  {
    for(cudf::size_type j = y; j < nrows; j += stride_y)
    {
      out_cols[j][i] = in_cols[i][j];
    }
  }
}

/**
 * @brief Transposes the validity mask
 * 
 * @param[in] in_cols_valid  pointers to the validity mask of the input columns
 * @param[out] out_cols_valid  pointers to the pre-allocated validity mask of
 *  the output columns
 * @param[out] out_cols_null_count  array of per output-row null counts
 * @param[in] ncols  number of columns in input table
 * @param[in] nrows  number of rows in input table
 */
__global__
void gpu_transpose_valids(cudf::valid_type **in_cols_valid,
                          cudf::valid_type **out_cols_valid,
                          cudf::size_type *out_cols_null_count,
                          cudf::size_type ncols, cudf::size_type nrows)
{
  using MaskType = uint32_t;
  constexpr uint32_t BITS_PER_MASK{sizeof(MaskType) * 8};

  cudf::size_type x = blockIdx.x * blockDim.x + threadIdx.x;
  cudf::size_type y = blockIdx.y * blockDim.y + threadIdx.y;

  cudf::size_type stride_x = blockDim.x * gridDim.x;
  cudf::size_type stride_y = blockDim.y * gridDim.y;

  cudf::size_type i = x;
  cudf::size_type j = y;
  auto active_threads = __ballot_sync(0xffffffff, i < ncols);
  while(i < ncols)
  {
    j = y;
    while(j < nrows)
    {
      bool const input_is_valid{gdf_is_valid(in_cols_valid[i], j)};
      MaskType const result_mask{__ballot_sync(active_threads, input_is_valid)};

      MaskType* const __restrict__ out_mask32 =
        reinterpret_cast<MaskType*>(out_cols_valid[j]);

      cudf::size_type const out_location = i / BITS_PER_MASK;

      // Only one thread writes output
      if (0 == threadIdx.x % warpSize) {
        out_mask32[out_location] = result_mask;
        int num_nulls = __popc(active_threads) - __popc(result_mask);
        atomicAdd(out_cols_null_count + j, num_nulls);
      }
      
      j += stride_y;
    }
    i += stride_x;
    active_threads = __ballot_sync(active_threads, i < ncols);
  }
}

// TODO: refactor and separate `valids` kernel launch into another function.
// Should not need to pass `has_null`
struct launch_kernel{
  template <typename ColumnType>
  gdf_error operator()(
    void **in_cols_data_ptr, void **out_cols_data_ptr,
    cudf::valid_type **in_cols_valid_ptr, cudf::valid_type **out_cols_valid_ptr,
    cudf::size_type *out_cols_nullct_ptr,
    cudf::size_type ncols, cudf::size_type nrows, bool has_null)
  {
    dim3 dimBlock(WARP_SIZE, WARP_SIZE, 1);
    dim3 dimGrid(std::min((ncols + WARP_SIZE - 1) / WARP_SIZE, MAX_GRID_SIZE),
                 std::min((nrows + WARP_SIZE - 1) / WARP_SIZE, MAX_GRID_SIZE),
                 1);

    gpu_transpose<ColumnType><<<dimGrid,dimBlock>>>(
      reinterpret_cast<ColumnType**>(in_cols_data_ptr),
      reinterpret_cast<ColumnType**>(out_cols_data_ptr),
      ncols, nrows
    );
    if (has_null){
      gpu_transpose_valids<<<dimGrid,dimBlock>>>(
        in_cols_valid_ptr,
        out_cols_valid_ptr,
        out_cols_nullct_ptr,
        ncols, nrows
      );
    }
    hipDeviceSynchronize();
    CUDA_CHECK_LAST();
    return GDF_SUCCESS;
  }
};

}

gdf_error gdf_transpose(cudf::size_type ncols, gdf_column** in_cols,
                        gdf_column** out_cols) {
  // Make sure the inputs are not null
  GDF_REQUIRE((ncols > 0) && (nullptr != in_cols) && (nullptr != out_cols),
              GDF_DATASET_EMPTY)

  // If there are no rows in the input, return successfully
  GDF_REQUIRE(in_cols[0]->size > 0, GDF_SUCCESS)

  // Check datatype homogeneity
  gdf_dtype dtype = in_cols[0]->dtype;
  for (cudf::size_type i = 1; i < ncols; i++) {
    GDF_REQUIRE(in_cols[i]->dtype == dtype, GDF_DTYPE_MISMATCH)
  }
  cudf::size_type nrows = in_cols[0]->size;
  cudf::size_type out_ncols = nrows;
  for (cudf::size_type i = 0; i < out_ncols; i++) {
    GDF_REQUIRE(out_cols[i]->dtype == dtype, GDF_DTYPE_MISMATCH)
  }

  // Check if there are nulls to be processed
  bool const has_null{ std::any_of(in_cols, in_cols + ncols, 
    [](gdf_column * col){ return col->null_count > 0; }) };

  if (has_null) {
    for (cudf::size_type i = 0; i < out_ncols; i++) {
      GDF_REQUIRE(out_cols[i]->valid != nullptr, GDF_VALIDITY_MISSING)
    }
  }

  cudf::nvtx::range_push("CUDF_TRANSPOSE", cudf::nvtx::color::GREEN);

  // Copy input columns `data` and `valid` pointers to device
  std::vector<void*> in_columns_data(ncols);
  std::vector<cudf::valid_type*> in_columns_valid(ncols);
  for (cudf::size_type i = 0; i < ncols; ++i) {
    in_columns_data[i] = in_cols[i]->data;
    in_columns_valid[i] = in_cols[i]->valid;
  }
  rmm::device_vector<void*> d_in_columns_data(in_columns_data);
  rmm::device_vector<cudf::valid_type*> d_in_columns_valid(in_columns_valid);

  void** in_cols_data_ptr = d_in_columns_data.data().get();
  cudf::valid_type** in_cols_valid_ptr = d_in_columns_valid.data().get();

  // Copy output columns `data` and `valid` pointers to device
  std::vector<void*> out_columns_data(out_ncols);
  std::vector<cudf::valid_type*> out_columns_valid(out_ncols);
  for (cudf::size_type i = 0; i < out_ncols; ++i) {
    out_columns_data[i] = out_cols[i]->data;
    out_columns_valid[i] = out_cols[i]->valid;
  }
  rmm::device_vector<void*> d_out_columns_data(out_columns_data);
  rmm::device_vector<cudf::valid_type*> d_out_columns_valid(out_columns_valid);
  rmm::device_vector<cudf::size_type> d_out_columns_nullct(out_ncols);

  void** out_cols_data_ptr = d_out_columns_data.data().get();
  cudf::valid_type** out_cols_valid_ptr = d_out_columns_valid.data().get();
  cudf::size_type* out_cols_nullct_ptr = d_out_columns_nullct.data().get();

  cudf::type_dispatcher(dtype,
                        launch_kernel{},
                        in_cols_data_ptr,
                        out_cols_data_ptr,
                        in_cols_valid_ptr,
                        out_cols_valid_ptr,
                        out_cols_nullct_ptr,
                        ncols, nrows, has_null);

  // Transfer null counts to gdf structs
  thrust::host_vector<cudf::size_type> out_columns_nullct(d_out_columns_nullct);
  for(cudf::size_type i = 0; i < out_ncols; i++)
  {
    out_cols[i]->null_count = out_columns_nullct[i];
  }
  
  cudf::nvtx::range_pop();
  return GDF_SUCCESS;
}