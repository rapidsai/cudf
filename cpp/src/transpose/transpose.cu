#include "hip/hip_runtime.h"
/* Copyright 2018 NVIDIA Corporation.  All rights reserved. */

#include <utilities/nvtx/nvtx_utils.h>
#include <utilities/type_dispatcher.hpp>
#include <rmm/thrust_rmm_allocator.h>
#include <bitmask/legacy/legacy_bitmask.hpp>
#include <cudf/cudf.h>
#include <hipcub/hipcub.hpp>
#include <memory>
#include <stdio.h>
#include <algorithm>

namespace
{

constexpr int WARP_SIZE = 32;
constexpr int MAX_GRID_SIZE = (1<<16)-1;

/**
 * @brief Transposes the values from ncols x nrows input columns to
 *  nrows x ncols output columns
 * 
 * @tparam ColumnType  Datatype of values pointed to by the pointers
 * @param in_cols[in]  Pointers to input columns' data
 * @param out_cols[out]  Pointers to pre-allocated output columns' data
 * @param ncols[in]  Number of columns in input table
 * @param nrows[in]  Number of rown in input table
 */
template <typename ColumnType>
__global__
void gpu_transpose(ColumnType **in_cols, ColumnType **out_cols,
                  gdf_size_type ncols, gdf_size_type nrows)
{
  gdf_size_type x = blockIdx.x * blockDim.x + threadIdx.x;
  gdf_size_type y = blockIdx.y * blockDim.y + threadIdx.y;
    
  gdf_size_type stride_x = blockDim.x * gridDim.x;
  gdf_size_type stride_y = blockDim.y * gridDim.y;

  for(gdf_size_type i = x; i < ncols; i += stride_x)
  {
    for(gdf_size_type j = y; j < nrows; j += stride_y)
    {
      out_cols[j][i] = in_cols[i][j];
    }
  }
}

/**
 * @brief Transposes the validity mask
 * 
 * @param[in] in_cols_valid  pointers to the validity mask of the input columns
 * @param[out] out_cols_valid  pointers to the pre-allocated validity mask of
 *  the output columns
 * @param[out] out_cols_null_count  array of per output-row null counts
 * @param[in] ncols  number of columns in input table
 * @param[in] nrows  number of rows in input table
 */
__global__
void gpu_transpose_valids(gdf_valid_type **in_cols_valid,
                          gdf_valid_type **out_cols_valid,
                          gdf_size_type *out_cols_null_count,
                          gdf_size_type ncols, gdf_size_type nrows)
{
  using MaskType = uint32_t;
  constexpr uint32_t BITS_PER_MASK{sizeof(MaskType) * 8};

  gdf_size_type x = blockIdx.x * blockDim.x + threadIdx.x;
  gdf_size_type y = blockIdx.y * blockDim.y + threadIdx.y;

  gdf_size_type stride_x = blockDim.x * gridDim.x;
  gdf_size_type stride_y = blockDim.y * gridDim.y;

  gdf_size_type i = x;
  gdf_size_type j = y;
  auto active_threads = __ballot_sync(0xffffffff, i < ncols);
  while(i < ncols)
  {
    j = y;
    while(j < nrows)
    {
      bool const input_is_valid{gdf_is_valid(in_cols_valid[i], j)};
      MaskType const result_mask{__ballot_sync(active_threads, input_is_valid)};

      MaskType* const __restrict__ out_mask32 =
        reinterpret_cast<MaskType*>(out_cols_valid[j]);

      gdf_index_type const out_location = i / BITS_PER_MASK;

      // Only one thread writes output
      if (0 == threadIdx.x % warpSize) {
        out_mask32[out_location] = result_mask;
        int num_nulls = __popc(active_threads) - __popc(result_mask);
        atomicAdd(out_cols_null_count + j, num_nulls);
      }
      
      j += stride_y;
    }
    i += stride_x;
    active_threads = __ballot_sync(active_threads, i < ncols);
  }
}

// TODO: refactor and separate `valids` kernel launch into another function.
// Should not need to pass `has_null`
struct launch_kernel{
  template <typename ColumnType>
  gdf_error operator()(
    void **in_cols_data_ptr, void **out_cols_data_ptr,
    gdf_valid_type **in_cols_valid_ptr, gdf_valid_type **out_cols_valid_ptr,
    gdf_size_type *out_cols_nullct_ptr,
    gdf_size_type ncols, gdf_size_type nrows, bool has_null)
  {
    dim3 dimBlock(WARP_SIZE, WARP_SIZE, 1);
    dim3 dimGrid(std::min((ncols + WARP_SIZE - 1) / WARP_SIZE, MAX_GRID_SIZE),
                 std::min((nrows + WARP_SIZE - 1) / WARP_SIZE, MAX_GRID_SIZE),
                 1);

    gpu_transpose<ColumnType><<<dimGrid,dimBlock>>>(
      reinterpret_cast<ColumnType**>(in_cols_data_ptr),
      reinterpret_cast<ColumnType**>(out_cols_data_ptr),
      ncols, nrows
    );
    if (has_null){
      gpu_transpose_valids<<<dimGrid,dimBlock>>>(
        in_cols_valid_ptr,
        out_cols_valid_ptr,
        out_cols_nullct_ptr,
        ncols, nrows
      );
    }
    hipDeviceSynchronize();
    CUDA_CHECK_LAST();
    return GDF_SUCCESS;
  }
};

}

gdf_error gdf_transpose(gdf_size_type ncols, gdf_column** in_cols,
                        gdf_column** out_cols) {
  // Make sure the inputs are not null
  GDF_REQUIRE((ncols > 0) && (nullptr != in_cols) && (nullptr != out_cols),
              GDF_DATASET_EMPTY)

  // If there are no rows in the input, return successfully
  GDF_REQUIRE(in_cols[0]->size > 0, GDF_SUCCESS)

  // Check datatype homogeneity
  gdf_dtype dtype = in_cols[0]->dtype;
  for (gdf_size_type i = 1; i < ncols; i++) {
    GDF_REQUIRE(in_cols[i]->dtype == dtype, GDF_DTYPE_MISMATCH)
  }
  gdf_size_type nrows = in_cols[0]->size;
  gdf_size_type out_ncols = nrows;
  for (gdf_size_type i = 0; i < out_ncols; i++) {
    GDF_REQUIRE(out_cols[i]->dtype == dtype, GDF_DTYPE_MISMATCH)
  }

  // Check if there are nulls to be processed
  bool const has_null{ std::any_of(in_cols, in_cols + ncols, 
    [](gdf_column * col){ return col->null_count > 0; }) };

  if (has_null) {
    for (gdf_size_type i = 0; i < out_ncols; i++) {
      GDF_REQUIRE(out_cols[i]->valid != nullptr, GDF_VALIDITY_MISSING)
    }
  }

  PUSH_RANGE("CUDF_TRANSPOSE", GDF_GREEN);

  // Copy input columns `data` and `valid` pointers to device
  std::vector<void*> in_columns_data(ncols);
  std::vector<gdf_valid_type*> in_columns_valid(ncols);
  for (gdf_size_type i = 0; i < ncols; ++i) {
    in_columns_data[i] = in_cols[i]->data;
    in_columns_valid[i] = in_cols[i]->valid;
  }
  rmm::device_vector<void*> d_in_columns_data(in_columns_data);
  rmm::device_vector<gdf_valid_type*> d_in_columns_valid(in_columns_valid);

  void** in_cols_data_ptr = d_in_columns_data.data().get();
  gdf_valid_type** in_cols_valid_ptr = d_in_columns_valid.data().get();

  // Copy output columns `data` and `valid` pointers to device
  std::vector<void*> out_columns_data(out_ncols);
  std::vector<gdf_valid_type*> out_columns_valid(out_ncols);
  for (gdf_size_type i = 0; i < out_ncols; ++i) {
    out_columns_data[i] = out_cols[i]->data;
    out_columns_valid[i] = out_cols[i]->valid;
  }
  rmm::device_vector<void*> d_out_columns_data(out_columns_data);
  rmm::device_vector<gdf_valid_type*> d_out_columns_valid(out_columns_valid);
  rmm::device_vector<gdf_size_type> d_out_columns_nullct(out_ncols);

  void** out_cols_data_ptr = d_out_columns_data.data().get();
  gdf_valid_type** out_cols_valid_ptr = d_out_columns_valid.data().get();
  gdf_size_type* out_cols_nullct_ptr = d_out_columns_nullct.data().get();

  cudf::type_dispatcher(dtype,
                        launch_kernel{},
                        in_cols_data_ptr,
                        out_cols_data_ptr,
                        in_cols_valid_ptr,
                        out_cols_valid_ptr,
                        out_cols_nullct_ptr,
                        ncols, nrows, has_null);

  // Transfer null counts to gdf structs
  thrust::host_vector<gdf_size_type> out_columns_nullct(d_out_columns_nullct);
  for(gdf_size_type i = 0; i < out_ncols; i++)
  {
    out_cols[i]->null_count = out_columns_nullct[i];
  }
  
  POP_RANGE();
  return GDF_SUCCESS;
}