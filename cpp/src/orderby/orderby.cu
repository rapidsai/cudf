/*
 * Copyright 2018-2019 BlazingDB, Inc.
 *     Copyright 2018 Jean Pierre Huaroto <jeanpierre@blazingdb.com>
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <type_traits>
#include <algorithm>

#include "cudf.h"
#include "utilities/cudf_utils.h"
#include "utilities/error_utils.h"

#include "rmm/thrust_rmm_allocator.h"

#include "../sqls/sqls_rtti_comp.h"

namespace{ //annonymus

  gdf_error multi_col_order_by(gdf_column** cols,
                               int8_t* asc_desc,
                               size_t ncols,
                               gdf_column* output_indices,
                               bool flag_nulls_are_smallest)
  {
    GDF_REQUIRE(cols != nullptr && output_indices != nullptr, GDF_DATASET_EMPTY);
    GDF_REQUIRE(cols[0]->size == output_indices->size, GDF_COLUMN_SIZE_MISMATCH);
    /* NOTE: providing support for indexes to be multiple different types explodes compilation time, such that it become infeasible */
    GDF_REQUIRE(output_indices->dtype == GDF_INT32, GDF_UNSUPPORTED_DTYPE);

    // Check for null so we can use a faster sorting comparator 
    bool const have_nulls{ std::any_of(cols, cols + ncols, [](gdf_column * col){ return col->null_count > 0; }) };

    rmm::device_vector<void*> d_cols(ncols);
    rmm::device_vector<gdf_valid_type*> d_valids(ncols);
    rmm::device_vector<int> d_types(ncols, 0);

    void** d_col_data = d_cols.data().get();
    gdf_valid_type** d_valids_data = d_valids.data().get();
    int* d_col_types = d_types.data().get();

    gdf_error gdf_status = soa_col_info(cols, ncols, d_col_data, d_valids_data, d_col_types);
    if(GDF_SUCCESS != gdf_status)
      return gdf_status;

		multi_col_sort(d_col_data, d_valids_data, d_col_types, asc_desc, ncols, cols[0]->size,
				have_nulls, static_cast<int32_t*>(output_indices->data), flag_nulls_are_smallest);

    return GDF_SUCCESS;
  }

} //end unknown namespace

/* --------------------------------------------------------------------------*/
/** 
 * @brief Sorts an array of gdf_column.
 * 
 * @Param[in] cols Array of gdf_columns
 * @Param[in] asc_desc Device array of sort order types for each column
 * (0 is ascending order and 1 is descending). If NULL is provided defaults
 * to ascending order for evey column.
 * @Param[in] ncols # columns
 * @Param[in] flag_nulls_are_smallest Flag to indicate if nulls are to be considered
 * smaller than non-nulls or viceversa
 * @Param[out] output_indices Pre-allocated gdf_column to be filled
 * with sorted indices
 * 
 * @Returns GDF_SUCCESS upon successful completion
 */
/* ----------------------------------------------------------------------------*/
gdf_error gdf_order_by(gdf_column** cols,
                       int8_t* asc_desc,
                       size_t ncols,
                       gdf_column* output_indices,
                       int flag_nulls_are_smallest)
{
  return multi_col_order_by(cols, asc_desc, ncols, output_indices, flag_nulls_are_smallest);
}
