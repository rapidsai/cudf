#include <cudf/cudf.h>
#include <rmm/rmm.h>
#include <utilities/cudf_utils.h>
#include <utilities/error_utils.hpp>
#include <cudf/utilities/legacy/type_dispatcher.hpp>
#include <cudf/utilities/legacy/wrapper_types.hpp>


#include <cub/device/device_segmented_radix_sort.cuh>


struct SegmentedRadixSortPlan{
    const gdf_size_type num_items;
    // temporary storage
    void *storage;
    size_t storage_bytes;
    void *back_key, *back_val;
    size_t back_key_size, back_val_size;

    hipStream_t stream;
    int descending;
    unsigned begin_bit, end_bit;

    SegmentedRadixSortPlan(size_t num_items, int descending,
                           unsigned begin_bit, unsigned end_bit)
        :   num_items(num_items),
            storage(nullptr), storage_bytes(0),
            back_key(nullptr), back_val(nullptr),
            back_key_size(0), back_val_size(0),
            stream(0), descending(descending),
            begin_bit(begin_bit), end_bit(end_bit)
    {}

    gdf_error setup(size_t sizeof_key, size_t sizeof_val) {
        back_key_size = num_items * sizeof_key;
        back_val_size = num_items * sizeof_val;
        RMM_TRY( RMM_ALLOC(&back_key, back_key_size, stream) ); // TODO: non-default stream
        RMM_TRY( RMM_ALLOC(&back_val, back_val_size, stream) );
        return GDF_SUCCESS;
    }

    gdf_error teardown() {
        RMM_TRY(RMM_FREE(back_key, stream));
        RMM_TRY(RMM_FREE(back_val, stream));
        RMM_TRY(RMM_FREE(storage, stream));
        return GDF_SUCCESS;
    }
};




template <typename Tk, typename Tv>
struct SegmentedRadixSort {

    static
    gdf_error sort( SegmentedRadixSortPlan *plan,
                    Tk *d_key_buf, Tv *d_value_buf,
                    unsigned num_segments,
                    unsigned *d_begin_offsets,
                    unsigned *d_end_offsets) {

        unsigned  num_items = plan->num_items;
        Tk *d_key_alt_buf = (Tk*)plan->back_key;
        Tv *d_value_alt_buf = (Tv*)plan->back_val;

        hipStream_t stream = plan->stream;
        int descending = plan->descending;
        unsigned begin_bit = plan->begin_bit;
        unsigned end_bit = plan->end_bit;

        hipcub::DoubleBuffer<Tk> d_keys(d_key_buf, d_key_alt_buf);

        typedef hipcub::DeviceSegmentedRadixSort Sorter;

        if (d_value_buf) {
            // Sort KeyValue pairs
            hipcub::DoubleBuffer<Tv> d_values(d_value_buf, d_value_alt_buf);
            if (descending) {
                Sorter::SortPairsDescending(plan->storage,
                                            plan->storage_bytes,
                                            d_keys,
                                            d_values,
                                            num_items,
                                            num_segments,
                                            d_begin_offsets,
                                            d_end_offsets,
                                            begin_bit,
                                            end_bit,
                                            stream);
            } else {
                Sorter::SortPairs(  plan->storage,
                                    plan->storage_bytes,
                                    d_keys,
                                    d_values,
                                    num_items,
                                    num_segments,
                                    d_begin_offsets,
                                    d_end_offsets,
                                    begin_bit,
                                    end_bit,
                                    stream    );
            }
            CUDA_CHECK_LAST();
            if (plan->storage && d_value_buf != d_values.Current()){
                hipMemcpyAsync(d_value_buf, d_value_alt_buf,
                                num_items * sizeof(Tv),
                                hipMemcpyDeviceToDevice,
                                stream);
                CUDA_CHECK_LAST();
            }
        } else {
            // Sort Keys only
            if (descending) {
                Sorter::SortKeysDescending(   plan->storage,
                                              plan->storage_bytes,
                                              d_keys,
                                              num_items,
                                              num_segments,
                                              d_begin_offsets,
                                              d_end_offsets,
                                              begin_bit,
                                              end_bit,
                                              stream  );
                CUDA_CHECK_LAST()

            } else {
                Sorter::SortKeys( plan->storage,
                                  plan->storage_bytes,
                                  d_keys,
                                  num_items,
                                  num_segments,
                                  d_begin_offsets,
                                  d_end_offsets,
                                  begin_bit,
                                  end_bit,
                                  stream  );
            }

            CUDA_CHECK_LAST();
        }

        if ( plan->storage ) {
            // We have operated and the result is not in front buffer
            if (d_key_buf != d_keys.Current()){
                hipMemcpyAsync(d_key_buf, d_key_alt_buf, num_items * sizeof(Tk),
                                          hipMemcpyDeviceToDevice, stream);
                CUDA_CHECK_LAST();
            }
        } else {
            // We have not operated.
            // Just checking for temporary storage requirement
            RMM_TRY( RMM_ALLOC(&plan->storage, plan->storage_bytes, plan->stream) ); // TODO: non-default stream
            CUDA_CHECK_LAST();
            // Now that we have allocated, do real work.
            return sort(plan, d_key_buf, d_value_buf, num_segments,
                        d_begin_offsets, d_end_offsets);
        }
        return GDF_SUCCESS;
    }
};


gdf_segmented_radixsort_plan_type* cffi_wrap(SegmentedRadixSortPlan* obj){
    return reinterpret_cast<gdf_segmented_radixsort_plan_type*>(obj);
}

SegmentedRadixSortPlan* cffi_unwrap(gdf_segmented_radixsort_plan_type* hdl){
    return reinterpret_cast<SegmentedRadixSortPlan*>(hdl);
}


gdf_segmented_radixsort_plan_type* gdf_segmented_radixsort_plan(
    size_t num_items, int descending,
    unsigned begin_bit, unsigned end_bit)
{
    return cffi_wrap(new SegmentedRadixSortPlan(num_items, descending,
    begin_bit, end_bit));
}

gdf_error gdf_segmented_radixsort_plan_setup(
    gdf_segmented_radixsort_plan_type *hdl,
    size_t sizeof_key, size_t sizeof_val)
{
    return cffi_unwrap(hdl)->setup(sizeof_key, sizeof_val);
}

gdf_error gdf_segmented_radixsort_plan_free(gdf_segmented_radixsort_plan_type *hdl)
{
    auto plan = cffi_unwrap(hdl);
    gdf_error status = plan->teardown();
    delete plan;
    return status;
}


template <typename Tv>
struct gdf_segmented_radixsort_functor
{
    template <typename Tk>
    gdf_error
    operator()( gdf_segmented_radixsort_plan_type *hdl,
                gdf_column *keycol,
                gdf_column *valcol,
                unsigned num_segments,
                unsigned *d_begin_offsets,
                unsigned *d_end_offsets)
    {
        /* validity mask must be empty */
        GDF_REQUIRE(!keycol->valid || !keycol->null_count, GDF_VALIDITY_UNSUPPORTED);
        GDF_REQUIRE(!valcol->valid || !valcol->null_count, GDF_VALIDITY_UNSUPPORTED);
        /* size of columns must match */
        GDF_REQUIRE(keycol->size == valcol->size, GDF_COLUMN_SIZE_MISMATCH);
        SegmentedRadixSortPlan *plan = cffi_unwrap(hdl);
        /* num_items must match */
        GDF_REQUIRE(plan->num_items == keycol->size, GDF_COLUMN_SIZE_MISMATCH);
        /* back buffer size must match */
        GDF_REQUIRE(sizeof(Tk) * plan->num_items == plan->back_key_size,
                    GDF_COLUMN_SIZE_MISMATCH);
        GDF_REQUIRE(sizeof(Tv) * plan->num_items == plan->back_val_size,
                    GDF_COLUMN_SIZE_MISMATCH);
        /* Do sort */
        return SegmentedRadixSort<Tk, Tv>::sort(plan,
                                    (Tk*)keycol->data, (Tv*)valcol->data,
                                        num_segments, d_begin_offsets, d_end_offsets);
    }
};

gdf_error gdf_segmented_radixsort(gdf_segmented_radixsort_plan_type *hdl,
                                  gdf_column *keycol,
                                  gdf_column *valcol,
                                  unsigned num_segments,
                                  unsigned *d_begin_offsets,
                                  unsigned *d_end_offsets)
{
    GDF_REQUIRE(valcol->dtype == GDF_INT64, GDF_UNSUPPORTED_DTYPE);

    return cudf::type_dispatcher(keycol->dtype,
                                gdf_segmented_radixsort_functor<int64_t>{},
                                hdl, keycol, valcol,
                                num_segments, d_begin_offsets,
                                d_end_offsets);
}


