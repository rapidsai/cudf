#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2025, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <cudf/column/column.hpp>
#include <cudf/column/column_view.hpp>
#include <cudf/utilities/type_dispatcher.hpp>

#include <rmm/cuda_stream_view.hpp>
#include <rmm/device_uvector.hpp>
#include <rmm/exec_policy.hpp>

#include <cub/device/device_radix_sort.cuh>

namespace cudf {
namespace detail {
namespace {

template <typename F>
struct float_pair {
  size_type s;
  F f;
};

template <typename F>
struct float_decomposer {
  __device__ cuda::std::tuple<size_type&, F&> operator()(float_pair<F>& key) const
  {
    return {key.s, key.f};
  }
};

template <typename F>
struct float_to_pair_fn {
  F const* fs;
  __device__ float_pair<F> operator()(cudf::size_type idx) const
  {
    auto const f = fs[idx];
    auto const s = (isnan(f) * (idx + 1));  // multiplier helps keep the sort stable for NaNs
    return float_pair<F>{s, f};
  }
};

struct faster_sort_fn {
  /**
   * @brief Sorts fixed-width columns using cub radix sort
   *
   * Should not be called if `input.has_nulls()==true`
   *
   * @param input Column to sort
   * @param output Output sorted column
   * @param ascending True if sort order is ascending
   * @param stream CUDA stream used for device memory operations and kernel launches
   */
  template <typename T>
  void faster_sort(column_view const& input,
                   mutable_column_view& output,
                   bool ascending,
                   rmm::cuda_stream_view stream)
  {
    auto d_in          = input.data<T>();
    auto d_out         = output.data<T>();
    auto const end_bit = sizeof(T) * 8;
    auto const sv      = stream.value();
    auto const n       = input.size();
    // cub radix sort implementation is always stable
    std::size_t tmp_bytes = 0;
    if (ascending) {
      hipcub::DeviceRadixSort::SortKeys(nullptr, tmp_bytes, d_in, d_out, n, 0, end_bit, sv);
      auto tmp_stg = rmm::device_buffer(tmp_bytes, stream);
      hipcub::DeviceRadixSort::SortKeys(tmp_stg.data(), tmp_bytes, d_in, d_out, n, 0, end_bit, sv);
    } else {
      hipcub::DeviceRadixSort::SortKeysDescending(nullptr, tmp_bytes, d_in, d_out, n, 0, end_bit, sv);
      auto tmp_stg = rmm::device_buffer(tmp_bytes, stream);
      hipcub::DeviceRadixSort::SortKeysDescending(
        tmp_stg.data(), tmp_bytes, d_in, d_out, n, 0, end_bit, sv);
    }
  }

  template <typename T>
  void operator()(column_view const& input,
                  mutable_column_view& output,
                  bool ascending,
                  rmm::cuda_stream_view stream)
    requires(cudf::is_floating_point<T>())
  {
    auto pair_in  = rmm::device_uvector<float_pair<T>>(input.size(), stream);
    auto d_in     = pair_in.begin();
    auto pair_out = rmm::device_uvector<float_pair<T>>(input.size(), stream);
    auto d_out    = pair_out.begin();

    thrust::transform(rmm::exec_policy_nosync(stream),
                      thrust::counting_iterator<size_type>(0),
                      thrust::counting_iterator<size_type>(input.size()),
                      d_in,
                      float_to_pair_fn<T>{input.begin<T>()});

    auto const decomposer = float_decomposer<T>{};
    auto const end_bit    = sizeof(float_pair<T>) * 8;
    auto const sv         = stream.value();
    auto const n          = input.size();
    // cub radix sort implementation is always stable
    std::size_t tmp_bytes = 0;
    if (ascending) {
      hipcub::DeviceRadixSort::SortKeys(
        nullptr, tmp_bytes, d_in, d_out, n, decomposer, 0, end_bit, sv);
      auto tmp_stg = rmm::device_buffer(tmp_bytes, stream);
      hipcub::DeviceRadixSort::SortKeys(
        tmp_stg.data(), tmp_bytes, d_in, d_out, n, decomposer, 0, end_bit, sv);
    } else {
      hipcub::DeviceRadixSort::SortKeysDescending(
        nullptr, tmp_bytes, d_in, d_out, n, decomposer, 0, end_bit, sv);
      auto tmp_stg = rmm::device_buffer(tmp_bytes, stream);
      hipcub::DeviceRadixSort::SortKeysDescending(
        tmp_stg.data(), tmp_bytes, d_in, d_out, n, decomposer, 0, end_bit, sv);
    }
    thrust::transform(rmm::exec_policy_nosync(stream),
                      d_out,
                      d_out + input.size(),
                      output.begin<T>(),
                      [] __device__(float_pair<T> const& p) { return p.f; });
  }

  template <typename T>
  void operator()(column_view const& input,
                  mutable_column_view& output,
                  bool ascending,
                  rmm::cuda_stream_view stream)
    requires(cudf::is_chrono<T>())
  {
    using rep_type = typename T::rep;
    faster_sort<rep_type>(input, output, ascending, stream);
  }

  template <typename T>
  void operator()(column_view const& input,
                  mutable_column_view& output,
                  bool ascending,
                  rmm::cuda_stream_view stream)
    requires(cudf::is_fixed_width<T>() and !cudf::is_chrono<T>() and !cudf::is_floating_point<T>())
  {
    faster_sort<T>(input, output, ascending, stream);
  }

  template <typename T>
  void operator()(column_view const&, mutable_column_view&, bool, rmm::cuda_stream_view)
    requires(not cudf::is_fixed_width<T>())
  {
    CUDF_UNREACHABLE("invalid type for faster sort");
  }
};

}  // namespace

bool is_radix_sortable(column_view const& column)
{
  return !column.has_nulls() && cudf::is_fixed_width(column.type());
}

std::unique_ptr<column> sort_radix(column_view const& input,
                                   bool ascending,
                                   rmm::cuda_stream_view stream,
                                   rmm::device_async_resource_ref mr)
{
  auto result   = std::make_unique<column>(input, stream, mr);
  auto out_view = result->mutable_view();
  cudf::type_dispatcher<dispatch_storage_type>(
    input.type(), faster_sort_fn{}, input, out_view, ascending, stream);
  return result;
}

}  // namespace detail
}  // namespace cudf
