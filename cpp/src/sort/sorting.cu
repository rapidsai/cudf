#include "cudf.h"
#include "rmm/rmm.h"
#include "utilities/cudf_utils.h"
#include "utilities/error_utils.hpp"
#include "utilities/type_dispatcher.hpp"
#include "utilities/wrapper_types.hpp"

#include <cub/device/device_radix_sort.cuh>

struct RadixSortPlan{
    const gdf_size_type num_items;
    // temporary storage
    void *storage;
    size_t storage_bytes;
    void *back_key, *back_val;
    size_t back_key_size, back_val_size;

    hipStream_t stream;
    int descending;
    unsigned begin_bit, end_bit;

    RadixSortPlan(size_t num_items, int descending,
                  unsigned begin_bit, unsigned end_bit)
        :   num_items(num_items),
            storage(nullptr), storage_bytes(0),
            back_key(nullptr), back_val(nullptr),
            back_key_size(0), back_val_size(0),
            stream(0), descending(descending),
            begin_bit(begin_bit), end_bit(end_bit)
    {}

    gdf_error setup(size_t sizeof_key, size_t sizeof_val) {
        back_key_size = num_items * sizeof_key;
        back_val_size = num_items * sizeof_val;
        RMM_TRY( RMM_ALLOC(&back_key, back_key_size, stream) ); // TODO: non-default stream
        RMM_TRY( RMM_ALLOC(&back_val, back_val_size, stream) );
        return GDF_SUCCESS;
    }

    gdf_error teardown() {
        RMM_TRY( RMM_FREE(back_key, stream) );
        RMM_TRY( RMM_FREE(back_val, stream) );
        RMM_TRY( RMM_FREE(storage, stream) );
        return GDF_SUCCESS;
    }
};


template <typename Tk, typename Tv>
struct RadixSort {

    static
    gdf_error sort( RadixSortPlan *plan, Tk *d_key_buf, Tv *d_value_buf) {

        unsigned  num_items = plan->num_items;
        Tk *d_key_alt_buf = (Tk*)plan->back_key;
        Tv *d_value_alt_buf = (Tv*)plan->back_val;

        hipStream_t stream = plan->stream;
        int descending = plan->descending;
        unsigned begin_bit = plan->begin_bit;
        unsigned end_bit = plan->end_bit;

        hipcub::DoubleBuffer<Tk> d_keys(d_key_buf, d_key_alt_buf);

        if (d_value_buf) {
            // Sort KeyValue pairs
            hipcub::DoubleBuffer<Tv> d_values(d_value_buf, d_value_alt_buf);
            if (descending) {
                hipcub::DeviceRadixSort::SortPairsDescending(plan->storage,
                                                          plan->storage_bytes,
                                                          d_keys,
                                                          d_values,
                                                          num_items,
                                                          begin_bit,
                                                          end_bit,
                                                          stream);
            } else {
                hipcub::DeviceRadixSort::SortPairs(  plan->storage,
                                                  plan->storage_bytes,
                                                  d_keys,
                                                  d_values,
                                                  num_items,
                                                  begin_bit,
                                                  end_bit,
                                                  stream    );
            }
            CUDA_CHECK_LAST();
            if (plan->storage && d_value_buf != d_values.Current()){
                hipMemcpyAsync(d_value_buf, d_value_alt_buf,
                                num_items * sizeof(Tv),
                                hipMemcpyDeviceToDevice,
                                stream);
                CUDA_CHECK_LAST();
            }
        } else {
            // Sort Keys only
            if (descending) {
                hipcub::DeviceRadixSort::SortKeysDescending(   plan->storage,
                                                            plan->storage_bytes,
                                                            d_keys,
                                                            num_items,
                                                            begin_bit,
                                                            end_bit,
                                                            stream  );
                CUDA_CHECK_LAST()

            } else {
                hipcub::DeviceRadixSort::SortKeys( plan->storage,
                                                plan->storage_bytes,
                                                d_keys,
                                                num_items,
                                                begin_bit,
                                                end_bit,
                                                stream  );
            }

            CUDA_CHECK_LAST();
        }

        if ( plan->storage ) {
            // We have operated and the result is not in front buffer
            if (d_key_buf != d_keys.Current()){
                hipMemcpyAsync(d_key_buf, d_key_alt_buf, num_items * sizeof(Tk),
                                hipMemcpyDeviceToDevice, stream);
                CUDA_CHECK_LAST();
            }
        } else {
            // We have not operated.
            // Just checking for temporary storage requirement
            RMM_TRY( RMM_ALLOC(&plan->storage, plan->storage_bytes, plan->stream) ); // TODO: non-default stream
            CUDA_CHECK_LAST();
            // Now that we have allocated, do real work.
            return sort(plan, d_key_buf, d_value_buf);
        }
        return GDF_SUCCESS;
    }
};

gdf_radixsort_plan_type* cffi_wrap(RadixSortPlan* obj){
    return reinterpret_cast<gdf_radixsort_plan_type*>(obj);
}

RadixSortPlan* cffi_unwrap(gdf_radixsort_plan_type* hdl){
    return reinterpret_cast<RadixSortPlan*>(hdl);
}


gdf_radixsort_plan_type* gdf_radixsort_plan(size_t num_items, int descending,
                                        unsigned begin_bit, unsigned end_bit){
    return cffi_wrap(new RadixSortPlan(num_items, descending,
                                       begin_bit, end_bit));
}

gdf_error gdf_radixsort_plan_setup(gdf_radixsort_plan_type *hdl,
                                   size_t sizeof_key,
                                   size_t sizeof_val)
{
    return cffi_unwrap(hdl)->setup(sizeof_key, sizeof_val);
}

gdf_error gdf_radixsort_plan_free(gdf_radixsort_plan_type *hdl) {
    auto plan = cffi_unwrap(hdl);
    gdf_error status = plan->teardown();
    delete plan;
    return status;
}

template <typename Tv>
struct gdf_radixsort_functor
{
    template <typename Tk>
    gdf_error
    operator()( gdf_radixsort_plan_type *hdl,
                gdf_column *keycol,
                gdf_column *valcol)
    {
        /* validity mask must be empty */
        GDF_REQUIRE(!keycol->valid || !keycol->null_count, GDF_VALIDITY_UNSUPPORTED);
        GDF_REQUIRE(!valcol->valid || !valcol->null_count, GDF_VALIDITY_UNSUPPORTED);
        /* size of columns must match */
        GDF_REQUIRE(keycol->size == valcol->size, GDF_COLUMN_SIZE_MISMATCH);
        RadixSortPlan *plan = cffi_unwrap(hdl);
        /* num_items must match */
        GDF_REQUIRE(plan->num_items == keycol->size, GDF_COLUMN_SIZE_MISMATCH);
        /* back buffer size must match */
        GDF_REQUIRE(sizeof(Tk) * plan->num_items == plan->back_key_size,
                    GDF_COLUMN_SIZE_MISMATCH);
        GDF_REQUIRE(sizeof(Tv) * plan->num_items == plan->back_val_size,
                    GDF_COLUMN_SIZE_MISMATCH);
        /* Do sort */
        return RadixSort<Tk, Tv>::sort(plan, (Tk*)keycol->data, (Tv*)valcol->data);
    }
};

gdf_error gdf_radixsort(gdf_radixsort_plan_type *hdl,
                        gdf_column *keycol,
                        gdf_column *valcol)
{
    GDF_REQUIRE(valcol->dtype == GDF_INT64, GDF_UNSUPPORTED_DTYPE);

    return cudf::type_dispatcher(keycol->dtype,
                                gdf_radixsort_functor<int64_t>{},
                                hdl, keycol, valcol);
}

