#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2020, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <cudf/column/column.hpp>
#include <cudf/column/column_factories.hpp>
#include <cudf/detail/null_mask.hpp>
#include <cudf/detail/sorting.hpp>
#include <cudf/sorting.hpp>
#include <cudf/table/row_operators.cuh>
#include <cudf/table/table.hpp>
#include <cudf/table/table_device_view.cuh>
#include <cudf/table/table_view.hpp>
#include <cudf/utilities/error.hpp>

#include <rmm/cuda_stream_view.hpp>
#include <rmm/device_vector.hpp>
#include <rmm/exec_policy.hpp>

#include <thrust/iterator/discard_iterator.h>
#include <thrust/iterator/permutation_iterator.h>
#include <thrust/sequence.h>

namespace cudf {
namespace detail {
namespace {
// Functor to identify unique elements in a sorted order table/column
template <bool has_nulls, typename ReturnType, typename Iterator>
struct unique_comparator {
  unique_comparator(table_device_view device_table, Iterator const sorted_order)
    : comparator(device_table, device_table, true), permute(sorted_order)
  {
  }
  __device__ ReturnType operator()(size_type index) const noexcept
  {
    return index == 0 || not comparator(permute[index], permute[index - 1]);
  };

 private:
  row_equality_comparator<has_nulls> comparator;
  Iterator const permute;
};

// Assign rank from 1 to n unique values. Equal values get same rank value.
rmm::device_vector<size_type> sorted_dense_rank(column_view input_col,
                                                column_view sorted_order_view,
                                                rmm::cuda_stream_view stream)
{
  auto device_table     = table_device_view::create(table_view{{input_col}}, stream);
  auto const input_size = input_col.size();
  rmm::device_vector<size_type> dense_rank_sorted(input_size);
  auto sorted_index_order = thrust::make_permutation_iterator(
    sorted_order_view.begin<size_type>(), thrust::make_counting_iterator<size_type>(0));
  if (input_col.has_nulls()) {
    auto conv = unique_comparator<true, size_type, decltype(sorted_index_order)>(
      *device_table, sorted_index_order);
    auto unique_it =
      thrust::make_transform_iterator(thrust::make_counting_iterator<size_type>(0), conv);

    thrust::inclusive_scan(
      rmm::exec_policy(stream), unique_it, unique_it + input_size, dense_rank_sorted.data().get());
  } else {
    auto conv = unique_comparator<false, size_type, decltype(sorted_index_order)>(
      *device_table, sorted_index_order);
    auto unique_it =
      thrust::make_transform_iterator(thrust::make_counting_iterator<size_type>(0), conv);

    thrust::inclusive_scan(
      rmm::exec_policy(stream), unique_it, unique_it + input_size, dense_rank_sorted.data().get());
  }
  return dense_rank_sorted;
}

/**
 * @brief Breaks the ties among equal value groups using binary operator and
 * transform this tied value to final rank.
 *
 * @param dense_rank dense rank of sorted input column (acts as key for value
 * groups).
 * @param tie_iter  iterator of rank to break ties among equal value groups.
 * @param sorted_order_view sorted order indices of input column
 * @param rank_iter output rank iterator
 * @param tie_breaker tie breaking operator. For example, maximum & minimum.
 * @param transformer transform after tie breaking (useful for average).
 * @param stream CUDA stream used for device memory operations and kernel launches.
 */
template <typename TieType,
          typename outputIterator,
          typename TieBreaker,
          typename Transformer,
          typename TieIterator>
void tie_break_ranks_transform(rmm::device_vector<size_type> const &dense_rank_sorted,
                               TieIterator tie_iter,
                               column_view const &sorted_order_view,
                               outputIterator rank_iter,
                               TieBreaker tie_breaker,
                               Transformer transformer,
                               rmm::cuda_stream_view stream)
{
  auto const input_size = sorted_order_view.size();
  rmm::device_vector<TieType> tie_sorted(input_size, 0);
  // algorithm: reduce_by_key(dense_rank, 1, n, reduction_tie_breaker)
  // reduction_tie_breaker = min, max, min_count
  thrust::reduce_by_key(rmm::exec_policy(stream),
                        dense_rank_sorted.begin(),
                        dense_rank_sorted.end(),
                        tie_iter,
                        thrust::make_discard_iterator(),
                        tie_sorted.begin(),
                        thrust::equal_to<size_type>{},
                        tie_breaker);
  auto sorted_tied_rank = thrust::make_transform_iterator(
    dense_rank_sorted.begin(),
    [tied_rank = tie_sorted.begin(), transformer] __device__(auto dense_pos) {
      return transformer(tied_rank[dense_pos - 1]);
    });
  thrust::scatter(rmm::exec_policy(stream),
                  sorted_tied_rank,
                  sorted_tied_rank + input_size,
                  sorted_order_view.begin<size_type>(),
                  rank_iter);
}

template <typename outputType>
void rank_first(column_view sorted_order_view,
                mutable_column_view rank_mutable_view,
                rmm::cuda_stream_view stream)
{
  // stable sort order ranking (no ties)
  thrust::scatter(rmm::exec_policy(stream),
                  thrust::make_counting_iterator<size_type>(1),
                  thrust::make_counting_iterator<size_type>(rank_mutable_view.size() + 1),
                  sorted_order_view.begin<size_type>(),
                  rank_mutable_view.begin<outputType>());
}

template <typename outputType>
void rank_dense(rmm::device_vector<size_type> const &dense_rank_sorted,
                column_view sorted_order_view,
                mutable_column_view rank_mutable_view,
                rmm::cuda_stream_view stream)
{
  // All equal values have same rank and rank always increases by 1 between groups
  thrust::scatter(rmm::exec_policy(stream),
                  dense_rank_sorted.begin(),
                  dense_rank_sorted.end(),
                  sorted_order_view.begin<size_type>(),
                  rank_mutable_view.begin<outputType>());
}

template <typename outputType>
void rank_min(rmm::device_vector<size_type> const &group_keys,
              column_view sorted_order_view,
              mutable_column_view rank_mutable_view,
              rmm::cuda_stream_view stream)
{
  // min of first in the group
  // All equal values have min of ranks among them.
  // algorithm: reduce_by_key(dense_rank, 1, n, min), scatter
  tie_break_ranks_transform<size_type>(group_keys,
                                       thrust::make_counting_iterator<size_type>(1),
                                       sorted_order_view,
                                       rank_mutable_view.begin<outputType>(),
                                       thrust::minimum<size_type>{},
                                       thrust::identity<outputType>{},
                                       stream);
}

template <typename outputType>
void rank_max(rmm::device_vector<size_type> const &group_keys,
              column_view sorted_order_view,
              mutable_column_view rank_mutable_view,
              rmm::cuda_stream_view stream)
{
  // max of first in the group
  // All equal values have max of ranks among them.
  // algorithm: reduce_by_key(dense_rank, 1, n, max), scatter
  tie_break_ranks_transform<size_type>(group_keys,
                                       thrust::make_counting_iterator<size_type>(1),
                                       sorted_order_view,
                                       rank_mutable_view.begin<outputType>(),
                                       thrust::maximum<size_type>{},
                                       thrust::identity<outputType>{},
                                       stream);
}

void rank_average(rmm::device_vector<size_type> const &group_keys,
                  column_view sorted_order_view,
                  mutable_column_view rank_mutable_view,
                  rmm::cuda_stream_view stream)
{
  // k, k+1, .. k+n-1
  // average = (n*k+ n*(n-1)/2)/n
  // average = k + (n-1)/2 = min + (count-1)/2
  // Calculate Min of ranks and Count of equal values
  // algorithm: reduce_by_key(dense_rank, 1, n, min_count)
  //            transform(min+(count-1)/2), scatter
  using MinCount = thrust::tuple<size_type, size_type>;
  tie_break_ranks_transform<MinCount>(
    group_keys,
    thrust::make_zip_iterator(thrust::make_tuple(thrust::make_counting_iterator<size_type>(1),
                                                 thrust::make_constant_iterator<size_type>(1))),
    sorted_order_view,
    rank_mutable_view.begin<double>(),
    [] __device__(auto rank_count1, auto rank_count2) {
      return MinCount{std::min(thrust::get<0>(rank_count1), thrust::get<0>(rank_count2)),
                      thrust::get<1>(rank_count1) + thrust::get<1>(rank_count2)};
    },
    [] __device__(MinCount minrank_count) {  // min+(count-1)/2
      return static_cast<double>(thrust::get<0>(minrank_count)) +
             (static_cast<double>(thrust::get<1>(minrank_count)) - 1) / 2.0;
    },
    stream);
}

}  // anonymous namespace

std::unique_ptr<column> rank(column_view const &input,
                             rank_method method,
                             order column_order,
                             null_policy null_handling,
                             null_order null_precedence,
                             bool percentage,
                             rmm::cuda_stream_view stream,
                             rmm::mr::device_memory_resource *mr)
{
  data_type const output_type = (percentage or method == rank_method::AVERAGE)
                                  ? data_type(type_id::FLOAT64)
                                  : data_type(type_to_id<size_type>());
  std::unique_ptr<column> rank_column = [&null_handling, &output_type, &input, &stream, &mr] {
    // na_option=keep assign NA to NA values
    if (null_handling == null_policy::EXCLUDE)
      return make_numeric_column(output_type,
                                 input.size(),
                                 detail::copy_bitmask(input, stream, mr),
                                 input.null_count(),
                                 stream,
                                 mr);
    else
      return make_numeric_column(output_type, input.size(), mask_state::UNALLOCATED, stream, mr);
  }();
  auto rank_mutable_view = rank_column->mutable_view();

  std::unique_ptr<column> sorted_order =
    (method == rank_method::FIRST)
      ? detail::stable_sorted_order(
          table_view{{input}}, {column_order}, {null_precedence}, stream, mr)
      : detail::sorted_order(table_view{{input}}, {column_order}, {null_precedence}, stream, mr);
  column_view sorted_order_view = sorted_order->view();

  // dense: All equal values have same rank and rank always increases by 1 between groups
  // acts as key for min, max, average to denote equal value groups
  rmm::device_vector<size_type> const dense_rank_sorted =
    [&method, &input, &sorted_order_view, &stream] {
      if (method != rank_method::FIRST)
        return sorted_dense_rank(input, sorted_order_view, stream);
      else
        return rmm::device_vector<size_type>();
    }();

  if (output_type.id() == type_id::FLOAT64) {
    switch (method) {
      case rank_method::FIRST:
        rank_first<double>(sorted_order_view, rank_mutable_view, stream);
        break;
      case rank_method::DENSE:
        rank_dense<double>(dense_rank_sorted, sorted_order_view, rank_mutable_view, stream);
        break;
      case rank_method::MIN:
        rank_min<double>(dense_rank_sorted, sorted_order_view, rank_mutable_view, stream);
        break;
      case rank_method::MAX:
        rank_max<double>(dense_rank_sorted, sorted_order_view, rank_mutable_view, stream);
        break;
      case rank_method::AVERAGE:
        rank_average(dense_rank_sorted, sorted_order_view, rank_mutable_view, stream);
        break;
      default: CUDF_FAIL("Unexpected rank_method for rank()");
    }
  } else {
    switch (method) {
      case rank_method::FIRST:
        rank_first<size_type>(sorted_order_view, rank_mutable_view, stream);
        break;
      case rank_method::DENSE:
        rank_dense<size_type>(dense_rank_sorted, sorted_order_view, rank_mutable_view, stream);
        break;
      case rank_method::MIN:
        rank_min<size_type>(dense_rank_sorted, sorted_order_view, rank_mutable_view, stream);
        break;
      case rank_method::MAX:
        rank_max<size_type>(dense_rank_sorted, sorted_order_view, rank_mutable_view, stream);
        break;
      case rank_method::AVERAGE:
        rank_average(dense_rank_sorted, sorted_order_view, rank_mutable_view, stream);
        break;
      default: CUDF_FAIL("Unexpected rank_method for rank()");
    }
  }

  // pct inplace transform
  if (percentage) {
    auto rank_iter = rank_mutable_view.begin<double>();
    size_type const count =
      (null_handling == null_policy::EXCLUDE) ? input.size() - input.null_count() : input.size();
    auto drs            = dense_rank_sorted.data().get();
    bool const is_dense = (method == rank_method::DENSE);
    thrust::transform(rmm::exec_policy(stream),
                      rank_iter,
                      rank_iter + input.size(),
                      rank_iter,
                      [is_dense, drs, count] __device__(double r) -> double {
                        return is_dense ? r / drs[count - 1] : r / count;
                      });
  }
  return rank_column;
}
}  // namespace detail

std::unique_ptr<column> rank(column_view const &input,
                             rank_method method,
                             order column_order,
                             null_policy null_handling,
                             null_order null_precedence,
                             bool percentage,
                             rmm::mr::device_memory_resource *mr)
{
  return detail::rank(input,
                      method,
                      column_order,
                      null_handling,
                      null_precedence,
                      percentage,
                      rmm::cuda_stream_default,
                      mr);
}
}  // namespace cudf
