/*
 * Copyright (c) 2021-2024, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "common_sort_impl.cuh"
#include "sort_column_impl.cuh"

#include <cudf/column/column_factories.hpp>
#include <cudf/column/column_view.hpp>
#include <cudf/utilities/type_dispatcher.hpp>

#include <rmm/resource_ref.hpp>

#include <thrust/sequence.h>

namespace cudf {
namespace detail {

/**
 * @copydoc
 * sorted_order(column_view&,order,null_order,rmm::cuda_stream_view,rmm::device_async_resource_ref )
 */
template <>
std::unique_ptr<column> sorted_order<sort_method::STABLE>(column_view const& input,
                                                          order column_order,
                                                          null_order null_precedence,
                                                          rmm::cuda_stream_view stream,
                                                          rmm::device_async_resource_ref mr)
{
  auto sorted_indices = cudf::make_numeric_column(
    data_type(type_to_id<size_type>()), input.size(), mask_state::UNALLOCATED, stream, mr);
  mutable_column_view indices_view = sorted_indices->mutable_view();
  thrust::sequence(
    rmm::exec_policy(stream), indices_view.begin<size_type>(), indices_view.end<size_type>(), 0);
  cudf::type_dispatcher<dispatch_storage_type>(input.type(),
                                               column_sorted_order_fn<sort_method::STABLE>{},
                                               input,
                                               indices_view,
                                               column_order == order::ASCENDING,
                                               null_precedence,
                                               stream);
  return sorted_indices;
}

}  // namespace detail
}  // namespace cudf
