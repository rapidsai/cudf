/*
 * Copyright (c) 2019-2021, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <cudf/column/column.hpp>
#include <cudf/detail/nvtx/ranges.hpp>
#include <cudf/detail/sorting.hpp>
#include <cudf/sorting.hpp>
#include <cudf/table/table_view.hpp>

#include <sort/sort_impl.cuh>

#include <rmm/cuda_stream_view.hpp>

namespace cudf {
namespace detail {
std::unique_ptr<column> sorted_order(table_view input,
                                     std::vector<order> const& column_order,
                                     std::vector<null_order> const& null_precedence,
                                     rmm::cuda_stream_view stream,
                                     rmm::mr::device_memory_resource* mr)
{
  return sorted_order<false>(input, column_order, null_precedence, stream, mr);
}

std::unique_ptr<table> sort_by_key(table_view const& values,
                                   table_view const& keys,
                                   std::vector<order> const& column_order,
                                   std::vector<null_order> const& null_precedence,
                                   rmm::cuda_stream_view stream,
                                   rmm::mr::device_memory_resource* mr)
{
  CUDF_EXPECTS(values.num_rows() == keys.num_rows(),
               "Mismatch in number of rows for values and keys");

  auto sorted_order = detail::sorted_order(keys, column_order, null_precedence, stream, mr);

  return detail::gather(values,
                        sorted_order->view(),
                        out_of_bounds_policy::DONT_CHECK,
                        detail::negative_index_policy::NOT_ALLOWED,
                        stream,
                        mr);
}

struct inplace_column_sort_fn {
  template <typename T, typename std::enable_if_t<cudf::is_fixed_width<T>()>* = nullptr>
  void operator()(mutable_column_view& col, bool ascending, rmm::cuda_stream_view stream) const
  {
    CUDF_EXPECTS(!col.has_nulls(), "Nulls not supported for in-place sort");
    using DeviceT = device_storage_type_t<T>;
    if (ascending) {
      thrust::sort(rmm::exec_policy(stream),
                   col.begin<DeviceT>(),
                   col.end<DeviceT>(),
                   thrust::less<DeviceT>());
    } else {
      thrust::sort(rmm::exec_policy(stream),
                   col.begin<DeviceT>(),
                   col.end<DeviceT>(),
                   thrust::greater<DeviceT>());
    }
  }

  template <typename T, typename std::enable_if_t<!cudf::is_fixed_width<T>()>* = nullptr>
  void operator()(mutable_column_view&, bool, rmm::cuda_stream_view) const
  {
    CUDF_FAIL("Column type must be relationally comparable and fixed-width");
  }
};

std::unique_ptr<table> sort(table_view input,
                            std::vector<order> const& column_order,
                            std::vector<null_order> const& null_precedence,
                            rmm::cuda_stream_view stream,
                            rmm::mr::device_memory_resource* mr)
{
  CUDF_FUNC_RANGE();
  // fast-path sort conditions: single, non-floating-point, fixed-width column with no nulls
  if (input.num_columns() == 1 && !input.column(0).has_nulls() &&
      cudf::is_fixed_width(input.column(0).type()) &&
      !cudf::is_floating_point(input.column(0).type())) {
    auto output    = std::make_unique<column>(input.column(0), stream, mr);
    auto view      = output->mutable_view();
    bool ascending = (column_order.empty() ? true : column_order.front() == order::ASCENDING);
    cudf::type_dispatcher(output->type(), inplace_column_sort_fn{}, view, ascending, stream);
    std::vector<std::unique_ptr<column>> columns;
    columns.emplace_back(std::move(output));
    return std::make_unique<table>(std::move(columns));
  }
  return detail::sort_by_key(
    input, input, column_order, null_precedence, rmm::cuda_stream_default, mr);
}

}  // namespace detail

std::unique_ptr<column> sorted_order(table_view input,
                                     std::vector<order> const& column_order,
                                     std::vector<null_order> const& null_precedence,
                                     rmm::mr::device_memory_resource* mr)
{
  CUDF_FUNC_RANGE();
  return detail::sorted_order(input, column_order, null_precedence, rmm::cuda_stream_default, mr);
}

std::unique_ptr<table> sort(table_view input,
                            std::vector<order> const& column_order,
                            std::vector<null_order> const& null_precedence,
                            rmm::mr::device_memory_resource* mr)
{
  CUDF_FUNC_RANGE();
  return detail::sort(input, column_order, null_precedence, rmm::cuda_stream_default, mr);
}

std::unique_ptr<table> sort_by_key(table_view const& values,
                                   table_view const& keys,
                                   std::vector<order> const& column_order,
                                   std::vector<null_order> const& null_precedence,
                                   rmm::mr::device_memory_resource* mr)
{
  CUDF_FUNC_RANGE();
  return detail::sort_by_key(
    values, keys, column_order, null_precedence, rmm::cuda_stream_default, mr);
}

}  // namespace cudf
