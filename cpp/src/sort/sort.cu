/*
 * Copyright (c) 2019-20, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "sort_impl.cuh"

#include <cudf/column/column.hpp>
#include <cudf/detail/nvtx/ranges.hpp>
#include <cudf/detail/sorting.hpp>
#include <cudf/sorting.hpp>
#include <cudf/table/table_view.hpp>

#include <rmm/cuda_stream_view.hpp>

namespace cudf {
namespace detail {
std::unique_ptr<column> sorted_order(table_view input,
                                     std::vector<order> const& column_order,
                                     std::vector<null_order> const& null_precedence,
                                     rmm::cuda_stream_view stream,
                                     rmm::mr::device_memory_resource* mr)
{
  return sorted_order<false>(input, column_order, null_precedence, stream, mr);
}

std::unique_ptr<table> sort_by_key(table_view const& values,
                                   table_view const& keys,
                                   std::vector<order> const& column_order,
                                   std::vector<null_order> const& null_precedence,
                                   rmm::cuda_stream_view stream,
                                   rmm::mr::device_memory_resource* mr)
{
  CUDF_EXPECTS(values.num_rows() == keys.num_rows(),
               "Mismatch in number of rows for values and keys");

  auto sorted_order = detail::sorted_order(keys, column_order, null_precedence, stream, mr);

  return detail::gather(values,
                        sorted_order->view(),
                        detail::out_of_bounds_policy::NULLIFY,
                        detail::negative_index_policy::NOT_ALLOWED,
                        stream,
                        mr);
}

}  // namespace detail

std::unique_ptr<column> sorted_order(table_view input,
                                     std::vector<order> const& column_order,
                                     std::vector<null_order> const& null_precedence,
                                     rmm::mr::device_memory_resource* mr)
{
  CUDF_FUNC_RANGE();
  return detail::sorted_order(input, column_order, null_precedence, rmm::cuda_stream_default, mr);
}

std::unique_ptr<table> sort(table_view input,
                            std::vector<order> const& column_order,
                            std::vector<null_order> const& null_precedence,
                            rmm::mr::device_memory_resource* mr)
{
  CUDF_FUNC_RANGE();
  return detail::sort_by_key(
    input, input, column_order, null_precedence, rmm::cuda_stream_default, mr);
}

std::unique_ptr<table> sort_by_key(table_view const& values,
                                   table_view const& keys,
                                   std::vector<order> const& column_order,
                                   std::vector<null_order> const& null_precedence,
                                   rmm::mr::device_memory_resource* mr)
{
  CUDF_FUNC_RANGE();
  return detail::sort_by_key(
    values, keys, column_order, null_precedence, rmm::cuda_stream_default, mr);
}

}  // namespace cudf
