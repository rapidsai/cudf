#include <cudf/cudf.h>


int gdf_cuda_last_error() {
    return hipGetLastError();
}

const char * gdf_cuda_error_string(int cuda_error) {
    return hipGetErrorString((hipError_t)cuda_error);
}

const char * gdf_cuda_error_name(int cuda_error) {
    return hipGetErrorName((hipError_t)cuda_error);
}
