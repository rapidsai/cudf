/*
 * Copyright (c) 2024, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <cudf/detail/utilities/cuda_memcpy.hpp>
#include <cudf/utilities/error.hpp>
#include <cudf/utilities/pinned_memory.hpp>

#include <rmm/exec_policy.hpp>

#include <thrust/copy.h>

namespace cudf::detail {

namespace {

void copy_pinned(void* dst, void const* src, std::size_t size, rmm::cuda_stream_view stream)
{
  if (size == 0) return;

  if (size < get_kernel_pinned_copy_threshold()) {
    thrust::copy_n(rmm::exec_policy_nosync(stream),
                   static_cast<const char*>(src),
                   size,
                   static_cast<char*>(dst));
  } else {
    CUDF_CUDA_TRY(hipMemcpyAsync(dst, src, size, hipMemcpyDefault, stream));
  }
}

void copy_pageable(void* dst, void const* src, std::size_t size, rmm::cuda_stream_view stream)
{
  if (size == 0) return;

  CUDF_CUDA_TRY(hipMemcpyAsync(dst, src, size, hipMemcpyDefault, stream));
}

};  // namespace

void cuda_memcpy_async(
  void* dst, void const* src, size_t size, host_memory_kind kind, rmm::cuda_stream_view stream)
{
  if (kind == host_memory_kind::PINNED) {
    copy_pinned(dst, src, size, stream);
  } else if (kind == host_memory_kind::PAGEABLE) {
    copy_pageable(dst, src, size, stream);
  } else {
    CUDF_FAIL("Unsupported host memory kind");
  }
}

void cuda_memcpy(
  void* dst, void const* src, size_t size, host_memory_kind kind, rmm::cuda_stream_view stream)
{
  cuda_memcpy_async(dst, src, size, kind, stream);
  stream.synchronize();
}

}  // namespace cudf::detail
