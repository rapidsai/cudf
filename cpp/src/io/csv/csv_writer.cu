#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
     * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <fstream>
#include <algorithm>
#include <hip/hip_runtime.h>
#include <nvstrings/NVStrings.h>
#include <rmm/rmm.h>
#include <rmm/thrust_rmm_allocator.h>
#include <cudf.h>
#include <utilities/error_utils.hpp>


//
// This is called by the write_csv method below.
//
// Parameters:
// - column:    The column to be converted.
// - delimiter: Separator to append to the column strings
// - null_representation: String to use for null entries
// - true_string: String to use for 'true' values in boolean columns
// - false_string: String to use for 'false' values in boolean columns
// Return: NVStrings instance formated for CSV column output.
//
NVStrings* column_to_strings_csv( gdf_column* column, const char* delimiter, const char* null_representation, const char* true_string, const char* false_string )
{
    NVStrings* rtn = nullptr;
    gdf_size_type rows = column->size;
    gdf_valid_type* valid = column->valid;
    switch( column->dtype )
    {
        case GDF_STRING:
            rtn = (static_cast<NVStrings*>(column->data))->copy();
            break;
        //case GDF_BOOL:
        //    rtn = NVStrings::create_from_bools((const bool*)col->data,rows,true_string,false_string,valid);
        //   break;
        case GDF_INT32:
            rtn = NVStrings::itos(static_cast<const int32_t*>(column->data),rows,valid);
            break;
        case GDF_INT64:
            rtn = NVStrings::ltos(static_cast<const int64_t*>(column->data),rows,valid);
            break;
        case GDF_FLOAT32:
            rtn = NVStrings::ftos(static_cast<const float*>(column->data),rows,valid);
            break;
        case GDF_FLOAT64:
            rtn = NVStrings::dtos(static_cast<const double*>(column->data),rows,valid);
            break;
        case GDF_DATE64:
            rtn = NVStrings::long2timestamp(static_cast<const uint64_t*>(column->data),rows,NVStrings::seconds,nullptr,valid);
            break;
        default:
            break;
    }
    CUDF_EXPECTS( rtn != nullptr, "write_csv: unsupported column type");

    // replace nulls if specified
    if( null_representation )
    {
        NVStrings* nstr = rtn->fillna(null_representation);
        NVStrings::destroy(rtn);
        rtn = nstr;
    }

    // probably could collapse this more
    bool bquoted = (column->dtype==GDF_STRING);
    // check for delimiters and quotes
    bool* bmatches = nullptr;
    RMM_TRY( RMM_ALLOC(&bmatches,rows*sizeof(bool),0) );
    if( rtn->contains("\"",bmatches) > 0 )
    {
        NVStrings* esc = rtn->replace("\"","\"\"");
        NVStrings::destroy(rtn);
        rtn = esc;
    }
    else if( rtn->contains(",",bmatches) > 0 )
        bquoted = true;
    RMM_TRY( RMM_FREE( bmatches, 0 ) );
    if( bquoted )
    {
        // prepend and append quotes if needed
        NVStrings* pre = rtn->slice_replace("\"",0,0);
        NVStrings::destroy(rtn);
        rtn = pre->slice_replace("\"",-1,-1);
        NVStrings::destroy(pre);
    }
    // append the delimiter last
    if( delimiter && *delimiter )
    {
        NVStrings* dstr = rtn->slice_replace(delimiter,-1,-1);
        NVStrings::destroy(rtn);
        rtn = dstr;
    }
    return rtn;
}

//---------------------------------------------------------------------------
// Creates CSV file from array of gdf_columns.
//
// This will create the CSV format by allocating host memory for the
// entire output and determine pointers for each row/column entry.
// Each column is converted to an NVStrings instance and then
// copied into their position in the output memory. This way,
// one column is processed at a time minimizing device memory usage.
//
//---------------------------------------------------------------------------
gdf_error write_csv(csv_write_arg* args)
{
    // when args becomes a struct/class these can be modified
    gdf_column** columns = args->columns;
    unsigned int count = (unsigned int)args->num_cols;
    gdf_size_type rows = columns[0]->size;
    const char* filepath = args->filepath;
    char delimiter[2] = {',','\0'};
    if( args->delimiter )
        delimiter[0] = args->delimiter;
    const char* terminator = "\n";
    if( args->line_terminator )
        terminator = args->line_terminator;
    const char* narep = args->na_rep;
    const char* true_value = (args->true_value ? args->true_value : "true");
    const char* false_value = (args->false_value ? args->false_value : "false");

    // check for issues here
    CUDF_EXPECTS( filepath!=nullptr, "write_csv: filepath not specified" );
    CUDF_EXPECTS( count!=0, "write_csv: num_cols is required" );
    CUDF_EXPECTS( columns!=0, "write_csv: invalid data values" );

    // check all columns are the same size
    const bool all_sizes_match = std::all_of( columns, columns+count,
        [rows] (gdf_column* col) {
            if( col->dtype==GDF_STRING )
            {
                NVStrings* strs = (NVStrings*)col->data;
                unsigned int elems = strs != nullptr ? strs->size() : 0;
                return (rows==(gdf_size_type)elems);
            }
            return (rows==col->size);
        });
    CUDF_EXPECTS( all_sizes_match, "write_csv: columns sizes do not match" );

    // check the file can be written
    std::ofstream filecsv(filepath,std::ios::out|std::ios::binary|std::ios::trunc);
    CUDF_EXPECTS( filecsv.is_open(), "write_csv: file could not be opened");

    //
    // It would be good if we could chunk this.
    // Use the rows*count calculation and a memory threshold to
    // output a subset of rows at a time instead of the whole thing at once.
    // The entire CSV must fit in CPU memory before writing it out.
    //
    // Compute string lengths for each string to go into the CSV output.
    std::unique_ptr<int[]> pstring_lengths(new int[rows*count]); // matrix of lengths
    int* string_lengths = pstring_lengths.get(); // each string length in each row,column
    size_t memsize = 0;
    for( unsigned int idx=0; idx < count; ++idx )
    {
        gdf_column* col = columns[idx];
        const char* delim = ((idx+1)<count ? delimiter : terminator);
        NVStrings* strs = column_to_strings_csv(col,delim,true_value,narep,false_value);
        memsize += strs->byte_count(string_lengths + (idx*rows),false);
        NVStrings::destroy(strs);
    }

    //
    // Example string_lengths matrix for 4 columns and 7 rows
    //                                     row-sums
    // col0:   1,  1,  2, 11, 12,  7,  7 |  41
    // col1:   1,  1,  2,  2,  3,  7,  6 |  22
    // col2:  20, 20, 20, 20, 20, 20, 20 | 140
    // col3:   5,  6,  4,  6,  4,  4,  5 |  34
    //        --------------------------------
    // col-   27, 28, 28, 39, 39, 38, 38 = 237   (for reference only)
    // sums
    //
    // Need to convert this into the following -- string_locations (below)
    //     0,  27,  55,  83, 122, 161, 199
    //     1,  28,  57,  94, 134, 168, 206
    //     2,  29,  59,  96, 137, 175, 212
    //    22,  49,  79, 116, 157, 195, 232
    //
    // This is essentially an exclusive-scan (prefix-sum) across columns.
    // Moving left-to-right, add up each column and carry each value to the next column.
    // Looks like we could transpose the matrix, scan it, and then untranspose it.
    // Should be able to parallelize the math for this -- will look at prefix-sum algorithms.
    //
    std::vector<char> buffer(memsize+1);
    std::vector<size_t> string_locations(rows*count); // all the memory pointers for each column
    string_locations[0] = 0; // first one is always 0
    // compute offsets as described above into locations matrix
    size_t offset = 0;
    for( gdf_size_type jdx=0; jdx < rows; ++jdx )
    {
        // add up column values for each row
        // this is essentially an exclusive-scan across columns
        string_locations[jdx] = (size_t)(buffer.data() + offset); // initialize first item
        for( unsigned int idx=0; idx < count; ++idx )
        {
            int* in = string_lengths + (idx*rows);
            int len = in[jdx];
            offset += (len > 0 ? len:0);
            if( (idx+1) < count )
            {
                size_t* out = string_locations.data() + ((idx+1)*rows);
                out[jdx] = (size_t)(buffer.data() + offset);
            }
        }
    }
    // now fill in the memory one column at a time
    for( unsigned int idx=0; idx < count; ++idx )
    {
        gdf_column* col = columns[idx];
        const char* delim = ((idx+1)<count ? delimiter : terminator);
        NVStrings* strs = column_to_strings_csv(col,delim,true_value,narep,false_value);
        size_t* colptrs = string_locations.data() + (idx*rows);
        // to_host places all the strings into their correct positions in host memory
        strs->to_host((char**)colptrs,0,rows);
        NVStrings::destroy(strs);
    }
    //buffer[memsize] = 0; // just so we can printf if needed
    // now write buffer to file
    // first write the header
    for( unsigned int idx=0; idx < count; ++idx )
    {
        gdf_column* col = columns[idx];
        const char* delim = ((idx+1)<count ? delimiter : terminator);
        if( col->col_name )
            filecsv << "\"" << col->col_name << "\"";
        filecsv << delim;
    }
    // now write the data
    filecsv.write(buffer.data(),memsize);
    filecsv.close();
    return gdf_error::GDF_SUCCESS;
}
