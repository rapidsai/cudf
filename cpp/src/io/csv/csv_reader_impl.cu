#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2018, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
	 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

/**
 * @file csv-reader.cu  code to read csv data
 *
 * CSV Reader
 */

#include "csv_gpu.h"
#include "csv_reader_impl.hpp"

#include <hip/hip_runtime.h>

#include <algorithm>
#include <iostream>
#include <numeric>
#include <string>
#include <vector>
#include <memory>
#include <unordered_map>
#include <cstring>

#include <stdio.h>
#include <stdlib.h>

#include <unistd.h>
#include <fcntl.h>
#include <sys/types.h>
#include <sys/stat.h>
#include <sys/mman.h>

#include <thrust/scan.h>
#include <thrust/reduce.h>
#include <thrust/device_ptr.h>
#include <thrust/execution_policy.h>

#include <thrust/host_vector.h>

#include "type_conversion.cuh"
#include "datetime_parser.cuh"

#include <cudf/cudf.h>
#include <utilities/error_utils.hpp>
#include <utilities/trie.cuh>
#include <cudf/utilities/legacy/type_dispatcher.hpp>
#include <utilities/cudf_utils.h> 

#include <nvstrings/NVStrings.h>

#include <rmm/rmm.h>
#include <rmm/thrust_rmm_allocator.h>
#include <io/comp/io_uncomp.h>

#include <io/cuio_common.hpp>
#include <io/utilities/datasource.hpp>
#include <io/utilities/parsing_utils.cuh>

using std::vector;
using std::string;

namespace cudf {
namespace io {
namespace csv {

/**---------------------------------------------------------------------------*
 * @brief Estimates the maximum expected length or a row, based on the number 
 * of columns
 * 
 * If the number of columns is not available, it will return a value large 
 * enough for most use cases
 * 
 * @param[in] num_columns Number of columns in the CSV file (optional)
 * 
 * @return Estimated maximum size of a row, in bytes
 *---------------------------------------------------------------------------**/
 constexpr size_t calculateMaxRowSize(int num_columns=0) noexcept {
	constexpr size_t max_row_bytes = 16*1024; // 16KB
	constexpr size_t column_bytes = 64;
	constexpr size_t base_padding = 1024; // 1KB
	if (num_columns == 0){
		// Use flat size if the number of columns is not known
		return max_row_bytes;
	}
	else {
		// Expand the size based on the number of columns, if available
		return base_padding + num_columns * column_bytes; 
	}
}

/**
 * @brief Translates a dtype string and returns its dtype enumeration and any 
 * extended dtype flags that are supported by cuIO. Often, this is a column
 * with the same underlying dtype the basic types, but with different parsing
 * interpretations.
 *
 * @param[in] dtype String containing the basic or extended dtype
 *
 * @return std::pair<gdf_dtype, column_parse::flags> Tuple of dtype and flags
 */
std::pair<gdf_dtype, column_parse::flags> get_dtype_info(
    const std::string &dtype) {
  if (dtype == "hex" || dtype == "hex64") {
    return std::make_pair(GDF_INT64, column_parse::as_hexadecimal);
  }
  if (dtype == "hex32") {
    return std::make_pair(GDF_INT32, column_parse::as_hexadecimal);
  }

  return std::make_pair(convertStringToDtype(dtype), column_parse::as_default);
}

/**
* @brief Removes the first and Last quote in the string
*/
string removeQuotes(string str, char quotechar) {
	// Exclude first and last quotation char
	const size_t first_quote = str.find(quotechar);
	if (first_quote != string::npos) {
		str.erase(first_quote, 1);
	}
	const size_t  last_quote = str.rfind(quotechar);
	if (last_quote != string::npos) {
		str.erase(last_quote, 1);
	}

	return str;
}

/**
 * @brief Parse the first row to set the column names in the raw_csv parameter 
 *
 * The first row can be either the header row, or the first data row
 *
 * @return void
*/
void reader::Impl::setColumnNamesFromCsv() {
	vector<char> first_row = header;
	// No header, read the first data row
	if (first_row.empty()) {
		uint64_t first_row_len{};
		// If file only contains one row, recStart[1] is not valid
		if (num_records > 1) {
			CUDA_TRY(hipMemcpy(&first_row_len, recStart.data() + 1, sizeof(uint64_t), hipMemcpyDefault));
		}
		else {
			// File has one row - use the file size for the row size
			first_row_len = num_bytes / sizeof(char);
		}
		first_row.resize(first_row_len);
		CUDA_TRY(hipMemcpy(first_row.data(), data.data(), first_row_len * sizeof(char), hipMemcpyDefault));
	}

	int num_cols = 0;

	bool quotation	= false;
	for (size_t pos = 0, prev = 0; pos < first_row.size(); ++pos) {
		// Flip the quotation flag if current character is a quotechar
		if(first_row[pos] == opts.quotechar) {
			quotation = !quotation;
		}
		// Check if end of a column/row
		else if (pos == first_row.size() - 1 ||
				 (!quotation && first_row[pos] == opts.terminator) ||
				 (!quotation && first_row[pos] == opts.delimiter)) {
			// This is the header, add the column name
			if (args_.header >= 0) {
				// Include the current character, in case the line is not terminated
				int col_name_len = pos - prev + 1;
				// Exclude the delimiter/terminator is present
				if (first_row[pos] == opts.delimiter || first_row[pos] == opts.terminator) {
					--col_name_len;
				}
				// Also exclude '\r' character at the end of the column name if it's part of the terminator
				if (col_name_len > 0 &&
					opts.terminator == '\n' &&
					first_row[pos] == '\n' &&
					first_row[pos - 1] == '\r') {
					--col_name_len;
				}

				const string new_col_name(first_row.data() + prev, col_name_len);
				col_names.push_back(removeQuotes(new_col_name, opts.quotechar));

				// Stop parsing when we hit the line terminator; relevant when there is a blank line following the header.
				// In this case, first_row includes multiple line terminators at the end, as the new recStart belongs
				// to a line that comes after the blank line(s)
				if (!quotation && first_row[pos] == opts.terminator){
					break;
				}
			}
			else {
				// This is the first data row, add the automatically generated name
				col_names.push_back(args_.prefix + std::to_string(num_cols));
			}
			num_cols++;

			// Skip adjacent delimiters if delim_whitespace is set
			while (opts.multi_delimiter &&
				   pos < first_row.size() &&
				   first_row[pos] == opts.delimiter && 
				   first_row[pos + 1] == opts.delimiter) {
				++pos;
			}
			prev = pos + 1;
		}
	}
}

/**---------------------------------------------------------------------------*
 * @brief Updates the object with the total number of rows and
 * quotation characters in the file
 *
 * Does not count the quotations if quotechar is set to '/0'.
 *
 * @param[in] h_data Pointer to the csv data in host memory
 * @param[in] h_size Size of the input data, in bytes
 *
 * @return void
 *---------------------------------------------------------------------------**/
void reader::Impl::countRecordsAndQuotes(const char *h_data, size_t h_size) {
	vector<char> chars_to_count{opts.terminator};
	if (opts.quotechar != '\0') {
		chars_to_count.push_back(opts.quotechar);
	}

	num_records = countAllFromSet(h_data, h_size, chars_to_count);

	// If not starting at an offset, add an extra row to account for the first row in the file
	if (byte_range_offset == 0) {
		++num_records;
	}
}

/**---------------------------------------------------------------------------*
 * @brief Updates the object with the offset of each row in the file
 * Also add positions of each quotation character in the file.
 *
 * Does not process the quotations if quotechar is set to '/0'.
 *
 * @param[in] h_data Pointer to the csv data in host memory
 * @param[in] h_size Size of the input data, in bytes
 *
 * @return void
 *---------------------------------------------------------------------------**/
void reader::Impl::setRecordStarts(const char *h_data, size_t h_size) {
	// Allocate space to hold the record starting points
	const bool last_line_terminated = (h_data[h_size - 1] == opts.terminator);
	// If the last line is not terminated, allocate space for the EOF entry (added later)
	const gdf_size_type record_start_count = num_records + (last_line_terminated ? 0 : 1);
	recStart = device_buffer<uint64_t>(record_start_count); 

	auto* find_result_ptr = recStart.data();
	if (byte_range_offset == 0) {
		find_result_ptr++;
		CUDA_TRY(hipMemsetAsync(recStart.data(), 0ull, sizeof(uint64_t)));
	}
	vector<char> chars_to_find{opts.terminator};
	if (opts.quotechar != '\0') {
		chars_to_find.push_back(opts.quotechar);
	}
	// Passing offset = 1 to return positions AFTER the found character
	findAllFromSet(h_data, h_size, chars_to_find, 1, find_result_ptr);

	// Previous call stores the record pinput_file.typeositions as encountered by all threads
	// Sort the record positions as subsequent processing may require filtering
	// certain rows or other processing on specific records
	thrust::sort(rmm::exec_policy()->on(0), recStart.data(), recStart.data() + num_records);

	// Currently, ignoring lineterminations within quotes is handled by recording
	// the records of both, and then filtering out the records that is a quotechar
	// or a linetermination within a quotechar pair. The future major refactoring
	// of reader and its kernels will probably use a different tactic.
	if (opts.quotechar != '\0') {
		vector<uint64_t> h_rec_starts(num_records);
		const size_t rec_start_size = sizeof(uint64_t) * (h_rec_starts.size());
		CUDA_TRY( hipMemcpy(h_rec_starts.data(), recStart.data(), rec_start_size, hipMemcpyDeviceToHost) );

		auto recCount = num_records;

		bool quotation = false;
		for (gdf_size_type i = 1; i < num_records; ++i) {
			if (h_data[h_rec_starts[i] - 1] == opts.quotechar) {
				quotation = !quotation;
				h_rec_starts[i] = num_bytes;
				recCount--;
			}
			else if (quotation) {
				h_rec_starts[i] = num_bytes;
				recCount--;
			}
		}

		CUDA_TRY( hipMemcpy(recStart.data(), h_rec_starts.data(), rec_start_size, hipMemcpyHostToDevice) );
		thrust::sort(rmm::exec_policy()->on(0), recStart.data(), recStart.data() + num_records);
		num_records = recCount;
	}

	if (!last_line_terminated){
		// Add the EOF as the last record when the terminator is missing in the last line
		const uint64_t eof_offset = h_size;
		CUDA_TRY(hipMemcpy(recStart.data() + num_records, &eof_offset, sizeof(uint64_t), hipMemcpyDefault));
		// Update the record count
		++num_records;
	}
}

/**---------------------------------------------------------------------------*
 * @brief Reads CSV-structured data and returns an array of gdf_columns.
 *
 * @return void
 *---------------------------------------------------------------------------**/
table reader::Impl::read()
{
	// TODO move initialization to constructor
	num_actual_cols = args_.names.size();
	num_active_cols = args_.names.size();

	if (args_.delim_whitespace) {
		opts.delimiter = ' ';
		opts.multi_delimiter = true;
	} else {
		opts.delimiter = args_.delimiter;
		opts.multi_delimiter = false;
	}
	opts.terminator = args_.lineterminator;
	if (args_.quotechar != '\0' && args_.quoting != QUOTE_NONE) {
		opts.quotechar = args_.quotechar;
		opts.keepquotes = false;
		opts.doublequote = args_.doublequote;
	} else {
		opts.quotechar = '\0';
		opts.keepquotes = true;
		opts.doublequote = false;
	}
	opts.skipblanklines = args_.skip_blank_lines;
	opts.comment = args_.comment;
	opts.dayfirst = args_.dayfirst;
	opts.decimal = args_.decimal;
	opts.thousands = args_.thousands;
	CUDF_EXPECTS(opts.decimal != opts.delimiter, "Decimal point cannot be the same as the delimiter");
	CUDF_EXPECTS(opts.thousands != opts.delimiter, "Thousands separator cannot be the same as the delimiter");

  const auto compression_type = inferCompressionType(
      args_.compression, args_.input_data_form, args_.filepath_or_buffer,
      {{"gz", "gzip"}, {"zip", "zip"}, {"bz2", "bz2"}, {"xz", "xz"}});

  if (byte_range_offset > 0 || byte_range_size > 0) {
    CUDF_EXPECTS(compression_type == "none",
                 "Compression unsupported when reading using byte range");
  }

	// Handle user-defined booleans values, whereby field data is substituted
	// with true/false values; CUDF booleans are int types of 0 or 1
	vector<string> true_values{"True", "TRUE", "true"};
	true_values.insert(true_values.end(), args_.true_values.begin(), args_.true_values.end());

	d_trueTrie = createSerializedTrie(true_values);
	opts.trueValuesTrie = d_trueTrie.data().get();

	vector<string> false_values{"False", "FALSE", "false"};
	false_values.insert(false_values.end(), args_.false_values.begin(), args_.false_values.end());
	d_falseTrie = createSerializedTrie(false_values);
	opts.falseValuesTrie = d_falseTrie.data().get();

	if (args_.na_filter && (args_.keep_default_na || !args_.na_values.empty())) {
		vector<string> na_values{
			"#N/A", "#N/A N/A", "#NA", "-1.#IND", 
			"-1.#QNAN", "-NaN", "-nan", "1.#IND", 
			"1.#QNAN", "N/A", "NA", "NULL", 
			"NaN", "n/a", "nan", "null"};
		if(!args_.keep_default_na){
			na_values.clear();
		}
		na_values.insert(na_values.end(), args_.na_values.begin(), args_.na_values.end());

		d_naTrie = createSerializedTrie(na_values);
		opts.naValuesTrie = d_naTrie.data().get();
	}

  size_t range_size = 0;
  if (byte_range_size != 0) {
    const auto num_columns = std::max(args_.names.size(), args_.dtype.size());
    range_size = byte_range_size + calculateMaxRowSize(num_columns);
  }

  auto source = [&] {
    if (args_.input_data_form == FILE_PATH) {
      return datasource::create(args_.filepath_or_buffer, byte_range_offset,
                                range_size);
    } else if (args_.input_data_form == HOST_BUFFER) {
      return datasource::create(args_.filepath_or_buffer.c_str(),
                                args_.filepath_or_buffer.size());
    } else {
      CUDF_FAIL("Invalid input type");
    }
  }();

  // Return an empty dataframe if no data and no column metadata to process
  if (source->empty() && (args_.names.empty() || args_.dtype.empty())) {
    return table();
  }

  // Transfer source data to GPU
  if (not source->empty()) {
    const char *h_uncomp_data = nullptr;
    size_t h_uncomp_size = 0;

    num_bytes = (range_size != 0) ? range_size : source->size();
    const auto buffer = source->get_buffer(byte_range_offset, num_bytes);

    std::vector<char> h_uncomp_data_owner;
    if (compression_type == "none") {
      // Do not use the owner vector here to avoid extra copy
      h_uncomp_data = reinterpret_cast<const char *>(buffer->data());
      h_uncomp_size = buffer->size();
    } else {
      CUDF_EXPECTS(
          getUncompressedHostData(
              reinterpret_cast<const char *>(buffer->data()), buffer->size(),
              compression_type, h_uncomp_data_owner) == GDF_SUCCESS,
          "Cannot decompress data");
      h_uncomp_data = h_uncomp_data_owner.data();
      h_uncomp_size = h_uncomp_data_owner.size();
    }

    countRecordsAndQuotes(h_uncomp_data, h_uncomp_size);
    setRecordStarts(h_uncomp_data, h_uncomp_size);
    uploadDataToDevice(h_uncomp_data, h_uncomp_size);
  }

	//-----------------------------------------------------------------------------
	//-- Populate the header

  // Check if the user gave us a list of column names
  if (not args_.names.empty()) {
    h_column_flags.resize(args_.names.size(), column_parse::enabled);
    col_names = args_.names;
  } else {
    setColumnNamesFromCsv();

    num_actual_cols = num_active_cols = col_names.size();

    h_column_flags.resize(num_actual_cols, column_parse::enabled);

    // Rename empty column names to "Unnamed: col_index"
    for (size_t col_idx = 0; col_idx < col_names.size(); ++col_idx) {
      if (col_names[col_idx].empty()) {
        col_names[col_idx] = string("Unnamed: ") + std::to_string(col_idx);
      }
    }

    // Looking for duplicates
    std::unordered_map<string, int> col_names_histogram;
    for (auto& col_name: col_names){
      // Operator [] inserts a default-initialized value if the given key is not present
      if (++col_names_histogram[col_name] > 1){
        if (args_.mangle_dupe_cols) {
          // Rename duplicates of column X as X.1, X.2, ...; First appearance stays as X
          col_name += "." + std::to_string(col_names_histogram[col_name] - 1);
        }
        else {
          // All duplicate columns will be ignored; First appearance is parsed
          const auto idx = &col_name - col_names.data();
          h_column_flags[idx] = column_parse::disabled;
        }
      }
    }

    // Update the number of columns to be processed, if some might have been removed
    if (!args_.mangle_dupe_cols) {
      num_active_cols = col_names_histogram.size();
    }
  }

  // User can specify which columns should be parsed
  if (not args_.use_cols_indexes.empty() || not args_.use_cols_names.empty()) {
    std::fill(h_column_flags.begin(), h_column_flags.end(), column_parse::disabled);

    for (const auto index : args_.use_cols_indexes) {
      h_column_flags[index] = column_parse::enabled;
    }
    num_active_cols = args_.use_cols_indexes.size();

    for (const auto name : args_.use_cols_names) {
      const auto it = std::find(col_names.begin(), col_names.end(), name);
      if (it != col_names.end()) {
        h_column_flags[it - col_names.begin()] = column_parse::enabled;
        num_active_cols++;
      }
    }
  }

  // User can specify which columns should be inferred as datetime
  if (not args_.infer_date_indexes.empty() || not args_.infer_date_names.empty()) {
    for (auto index : args_.infer_date_indexes) {
      h_column_flags[index] |= column_parse::as_datetime;
    }
    for (auto name : args_.infer_date_names) {
      auto it = std::find(col_names.begin(), col_names.end(), name);
      if (it != col_names.end()) {
        h_column_flags[it - col_names.begin()] |= column_parse::as_datetime;
      }
    }
  }

  const std::vector<gdf_dtype> dtypes = gather_column_dtypes();

  // Alloc output; columns' data memory is still expected for empty dataframe
  std::vector<gdf_column_wrapper> columns;
  for (int col = 0, active_col = 0; col < num_actual_cols; ++col) {
    if (h_column_flags[col] & column_parse::enabled) {
      auto time_unit = TIME_UNIT_NONE;
      if (dtypes[active_col] == GDF_DATE64 || dtypes[active_col] == GDF_TIMESTAMP) {
        time_unit = TIME_UNIT_ms;
      }
      columns.emplace_back(num_records, dtypes[active_col],
                           gdf_dtype_extra_info{time_unit},
                           col_names[col]);
      columns.back().allocate();
      active_col++;
    }
  }

  // Convert CSV input to cuDF output
  if (num_records != 0) {
    decode_data(columns);
  }

  for (int i = 0; i < num_active_cols; ++i) {
    if (columns[i]->dtype == GDF_STRING) {
      using str_pair = std::pair<const char *, size_t>;
      using str_ptr = std::unique_ptr<NVStrings, decltype(&NVStrings::destroy)>;

      auto str_list = static_cast<str_pair *>(columns[i]->data);
      str_ptr str_data(NVStrings::create_from_index(str_list, columns[i]->size),
                       &NVStrings::destroy);
      CUDF_EXPECTS(str_data != nullptr, "Cannot create `NvStrings` instance");
      RMM_TRY(RMM_FREE(columns[i]->data, 0));

      // PANDAS' default behavior of enabling doublequote for two consecutive
      // quotechars in quoted fields results in reduction to a single quotechar
      if ((opts.quotechar != '\0') && (opts.doublequote == true)) {
        const std::string quotechar(1, opts.quotechar);
        const std::string doublequotechar(2, opts.quotechar);
        columns[i]->data =
            str_data->replace(doublequotechar.c_str(), quotechar.c_str());
      } else {
        columns[i]->data = str_data.release();
      }
    }
  }

  // Transfer ownership to raw pointer output arguments
  std::vector<gdf_column *> out_cols(columns.size());
  for (size_t i = 0; i < columns.size(); ++i) {
    out_cols[i] = columns[i].release();
  }

  return cudf::table(out_cols.data(), out_cols.size());
}

/**---------------------------------------------------------------------------*
 * @brief Uploads the relevant segment of the input csv data onto the GPU.
 * 
 * Only rows that need to be read are copied to the GPU, based on parameters
 * like nrows, skipheader, skipfooter.
 * Also updates the array of record starts to match the device data offset.
 * 
 * @param[in] h_uncomp_data Pointer to the uncompressed csv data in host memory
 * @param[in] h_uncomp_size Size of the input data, in bytes
 * 
 * @return void
 *---------------------------------------------------------------------------**/
void reader::Impl::uploadDataToDevice(const char *h_uncomp_data, size_t h_uncomp_size) {

  // Exclude the rows that are to be skipped from the start
  CUDF_EXPECTS(num_records > skiprows, "Skipping too many rows");
  const auto first_row = skiprows;
  num_records = num_records - first_row;

  std::vector<uint64_t> h_rec_starts(num_records);
  CUDA_TRY(hipMemcpy(h_rec_starts.data(), recStart.data() + first_row,
                      sizeof(uint64_t) * h_rec_starts.size(),
                      hipMemcpyDefault));

  // Trim lines that are outside range, but keep one greater for the end offset
  if (byte_range_size != 0) {
    auto it = h_rec_starts.end() - 1;
    while (it >= h_rec_starts.begin() &&
           *it > uint64_t(byte_range_size)) {
      --it;
    }
    if ((it + 2) < h_rec_starts.end()) {
      h_rec_starts.erase(it + 2, h_rec_starts.end());
    }
  }

  // Discard only blank lines, only fully comment lines, or both.
  // If only handling one of them, ensure it doesn't match against \0 as we do
  // not want certain scenarios to be filtered out (end-of-file)
  if (opts.skipblanklines || opts.comment != '\0') {
    const auto match_newline = opts.skipblanklines ? opts.terminator
                                                            : opts.comment;
    const auto match_comment = opts.comment != '\0' ? opts.comment
                                                             : match_newline;
    const auto match_return = (opts.skipblanklines &&
                              opts.terminator == '\n') ? '\r'
                                                                : match_comment;
    h_rec_starts.erase(
        std::remove_if(h_rec_starts.begin(), h_rec_starts.end(),
                       [&](uint64_t i) {
                         return (h_uncomp_data[i] == match_newline ||
                                 h_uncomp_data[i] == match_return ||
                                 h_uncomp_data[i] == match_comment);
                       }),
        h_rec_starts.end());
  }

  num_records = h_rec_starts.size();

  // Exclude the rows before the header row (inclusive)
  // But copy the header data for parsing the column names later (if necessary)
  if (args_.header >= 0) {
    header.assign(
        h_uncomp_data + h_rec_starts[args_.header],
        h_uncomp_data + h_rec_starts[args_.header + 1]);
    h_rec_starts.erase(h_rec_starts.begin(),
                       h_rec_starts.begin() + args_.header + 1);
    num_records = h_rec_starts.size();
  }

  // Exclude the rows that exceed past the requested number
  if (nrows >= 0 && nrows < num_records) {
    h_rec_starts.resize(nrows + 1);    // include end offset
    num_records = h_rec_starts.size();
  }

  // Exclude the rows that are to be skipped from the end
  if (skipfooter > 0) {
    h_rec_starts.resize(h_rec_starts.size() - skipfooter);
    num_records = h_rec_starts.size();
  }

  CUDF_EXPECTS(num_records > 0, "No data available for parsing");

  const auto start_offset = h_rec_starts.front();
  const auto end_offset = h_rec_starts.back();
  num_bytes = end_offset - start_offset;
  assert(num_bytes <= h_uncomp_size);
  num_bits = (num_bytes + 63) / 64;

  // Resize and upload the rows of interest
  recStart.resize(num_records);
  CUDA_TRY(hipMemcpy(recStart.data(), h_rec_starts.data(),
                      sizeof(uint64_t) * num_records,
                      hipMemcpyDefault));

  // Upload the raw data that is within the rows of interest
  data = device_buffer<char>(num_bytes);
  CUDA_TRY(hipMemcpy(data.data(), h_uncomp_data + start_offset,
                      num_bytes, hipMemcpyHostToDevice));

  // Adjust row start positions to account for the data subcopy
  thrust::transform(rmm::exec_policy()->on(0), recStart.data(),
                    recStart.data() + num_records,
                    thrust::make_constant_iterator(start_offset),
                    recStart.data(), thrust::minus<uint64_t>());

  // The array of row offsets includes EOF
  // reduce the number of records by one to exclude it from the row count
  num_records--;
}

std::vector<gdf_dtype> reader::Impl::gather_column_dtypes() {
  std::vector<gdf_dtype> dtypes;

  if (args_.dtype.empty()) {
    if (num_records == 0) {
      dtypes.resize(num_active_cols, GDF_STRING);
    } else {
      d_column_flags = h_column_flags;

      hostdevice_vector<column_parse::stats> column_stats(num_active_cols);
      CUDA_TRY(hipMemsetAsync(column_stats.device_ptr(), 0,
                               column_stats.memory_size()));
      CUDA_TRY(gpu::DetectCsvDataTypes(
          data.data(), recStart.data(), num_records, num_actual_cols, opts,
          d_column_flags.data().get(), column_stats.device_ptr()));
      CUDA_TRY(
          hipMemcpyAsync(column_stats.host_ptr(), column_stats.device_ptr(),
                          column_stats.memory_size(), hipMemcpyDeviceToHost));
      CUDA_TRY(hipStreamSynchronize(0));

      for (int col = 0; col < num_active_cols; col++) {
        unsigned long long countInt =
            column_stats[col].countInt8 + column_stats[col].countInt16 +
            column_stats[col].countInt32 + column_stats[col].countInt64;

        if (column_stats[col].countNULL == num_records) {
          // Entire column is NULL; allocate the smallest amount of memory
          dtypes.push_back(GDF_INT8);
        } else if (column_stats[col].countString > 0L) {
          dtypes.push_back(GDF_STRING);
        } else if (column_stats[col].countDateAndTime > 0L) {
          dtypes.push_back(GDF_DATE64);
        } else if (column_stats[col].countBool > 0L) {
          dtypes.push_back(GDF_BOOL8);
        } else if (column_stats[col].countFloat > 0L ||
                   (column_stats[col].countFloat == 0L && countInt > 0L &&
                    column_stats[col].countNULL > 0L)) {
          // The second condition has been added to conform to
          // PANDAS which states that a column of integers with
          // a single NULL record need to be treated as floats.
          dtypes.push_back(GDF_FLOAT64);
        } else {
          // All other integers are stored as 64-bit to conform to PANDAS
          dtypes.push_back(GDF_INT64);
        }
      }
    }
  } else {
    const bool is_dict = std::all_of(
        args_.dtype.begin(), args_.dtype.end(),
        [](const auto &s) { return s.find(':') != std::string::npos; });

    if (!is_dict) {
      if (args_.dtype.size() == 1) {
        // If it's a single dtype, assign that dtype to all active columns
        const auto dtype_info = get_dtype_info(args_.dtype[0]);
        dtypes.resize(num_active_cols, dtype_info.first);
        for (int col = 0; col < num_actual_cols; col++) {
          h_column_flags[col] |= dtype_info.second;
        }
        CUDF_EXPECTS(dtypes.back() != GDF_invalid, "Unsupported data type");
      } else {
        // If it's a list, assign dtypes to active columns in the given order
        CUDF_EXPECTS(static_cast<int>(args_.dtype.size()) >= num_actual_cols,
                     "Must specify data types for all columns");
        for (int col = 0; col < num_actual_cols; col++) {
          if (h_column_flags[col] & column_parse::enabled) {
            const auto dtype_info = get_dtype_info(args_.dtype[col]);
            dtypes.push_back(dtype_info.first);
            h_column_flags[col] |= dtype_info.second;
            CUDF_EXPECTS(dtypes.back() != GDF_invalid, "Unsupported data type");
          }
        }
      }
    } else {
      // Translate vector of `name : dtype` strings to map
      // NOTE: Incoming pairs can be out-of-order from column names in dataset
      std::unordered_map<std::string, std::string> col_type_map;
      for (const auto &pair : args_.dtype) {
        const auto pos = pair.find_last_of(':');
        const auto name = pair.substr(0, pos);
        const auto dtype = pair.substr(pos + 1, pair.size());
        col_type_map[name] = dtype;
      }

      for (int col = 0; col < num_actual_cols; col++) {
        if (h_column_flags[col] & column_parse::enabled) {
          CUDF_EXPECTS(col_type_map.find(col_names[col]) != col_type_map.end(),
                       "Must specify data types for all active columns");
          const auto dtype_info = get_dtype_info(col_type_map[col_names[col]]);
          dtypes.push_back(dtype_info.first);
          h_column_flags[col] |= dtype_info.second;
          CUDF_EXPECTS(dtypes.back() != GDF_invalid, "Unsupported data type");
        }
      }
    }
  }

  return dtypes;
}

void reader::Impl::decode_data(const std::vector<gdf_column_wrapper> &columns) {
  thrust::host_vector<gdf_dtype> h_dtypes(num_active_cols);
  thrust::host_vector<void*> h_data(num_active_cols);
  thrust::host_vector<gdf_valid_type*> h_valid(num_active_cols);

  for (int i = 0; i < num_active_cols; ++i) {
    h_dtypes[i] = columns[i]->dtype;
    h_data[i] = columns[i]->data;
    h_valid[i] = columns[i]->valid;
  }

  rmm::device_vector<gdf_dtype> d_dtypes = h_dtypes;
  rmm::device_vector<void*> d_data = h_data;
  rmm::device_vector<gdf_valid_type*> d_valid = h_valid;
  rmm::device_vector<gdf_size_type> d_valid_counts(num_active_cols, 0);
  d_column_flags = h_column_flags;

  CUDA_TRY(gpu::DecodeCsvColumnData(
      data.data(), recStart.data(), num_records, num_actual_cols, opts,
      d_column_flags.data().get(), d_dtypes.data().get(), d_data.data().get(),
      d_valid.data().get(), d_valid_counts.data().get()));
  CUDA_TRY(hipStreamSynchronize(0));

  thrust::host_vector<gdf_size_type> h_valid_counts = d_valid_counts;
  for (int i = 0; i < num_active_cols; ++i) {
    columns[i]->null_count = columns[i]->size - h_valid_counts[i];
  }
}

reader::Impl::Impl(reader_options const &args) : args_(args) {}

table reader::Impl::read_byte_range(size_t offset, size_t size) {
  byte_range_offset = offset;
  byte_range_size = size;
  return read();
}

table reader::Impl::read_rows(gdf_size_type num_skip_header,
                              gdf_size_type num_skip_footer,
                              gdf_size_type num_rows) {
  CUDF_EXPECTS(num_rows == -1 || num_skip_footer == 0,
               "cannot use both num_rows and num_skip_footer parameters");

  skiprows = num_skip_header;
  nrows = num_rows;
  skipfooter = num_skip_footer;
  return read();
}

reader::reader(reader_options const &args)
    : impl_(std::make_unique<Impl>(args)) {}

table reader::read() { return impl_->read(); }

table reader::read_byte_range(size_t offset, size_t size) {
  return impl_->read_byte_range(offset, size);
}
table reader::read_rows(gdf_size_type num_skip_header,
                        gdf_size_type num_skip_footer, gdf_size_type num_rows) {
  return impl_->read_rows(num_skip_header, num_skip_footer, num_rows);
}

reader::~reader() = default;

} // namespace csv
} // namespace io
} // namespace cudf
