#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2018, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
	 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

/**
 * @file csv-reader.cu  code to read csv data
 *
 * CSV Reader
 */

#include "csv_reader_impl.hpp"

#include <hip/hip_runtime.h>

#include <algorithm>
#include <iostream>
#include <numeric>
#include <string>
#include <vector>
#include <memory>
#include <unordered_map>
#include <cstring>

#include <stdio.h>
#include <stdlib.h>

#include <unistd.h>
#include <fcntl.h>
#include <sys/types.h>
#include <sys/stat.h>
#include <sys/mman.h>

#include <thrust/scan.h>
#include <thrust/reduce.h>
#include <thrust/device_ptr.h>
#include <thrust/execution_policy.h>

#include <thrust/host_vector.h>

#include "type_conversion.cuh"
#include "datetime_parser.cuh"

#include <cudf/cudf.h>
#include <utilities/error_utils.hpp>
#include <utilities/trie.cuh>
#include <utilities/type_dispatcher.hpp>
#include <utilities/cudf_utils.h> 

#include <nvstrings/NVStrings.h>

#include <rmm/rmm.h>
#include <rmm/thrust_rmm_allocator.h>
#include <io/comp/io_uncomp.h>

#include <io/cuio_common.hpp>
#include <io/utilities/datasource.hpp>
#include <io/utilities/parsing_utils.cuh>

using std::vector;
using std::string;

namespace cudf {
namespace io {
namespace csv {

using string_pair = std::pair<const char*,size_t>;

__global__ void convertCsvToGdf(char *csv, const ParseOptions opts,
                                gdf_size_type num_records, int num_columns,
                                column_parse::flags *flags, uint64_t *recStart,
                                gdf_dtype *dtype, void **data,
                                gdf_valid_type **valid,
                                gdf_size_type *num_valid);
__global__ void dataTypeDetection(char *raw_csv, const ParseOptions opts,
                                  gdf_size_type num_records, int num_columns,
                                  column_parse::flags *flags,
                                  uint64_t *recStart,
                                  column_data_t *d_columnData);

/**---------------------------------------------------------------------------*
 * @brief Estimates the maximum expected length or a row, based on the number 
 * of columns
 * 
 * If the number of columns is not available, it will return a value large 
 * enough for most use cases
 * 
 * @param[in] num_columns Number of columns in the CSV file (optional)
 * 
 * @return Estimated maximum size of a row, in bytes
 *---------------------------------------------------------------------------**/
 constexpr size_t calculateMaxRowSize(int num_columns=0) noexcept {
	constexpr size_t max_row_bytes = 16*1024; // 16KB
	constexpr size_t column_bytes = 64;
	constexpr size_t base_padding = 1024; // 1KB
	if (num_columns == 0){
		// Use flat size if the number of columns is not known
		return max_row_bytes;
	}
	else {
		// Expand the size based on the number of columns, if available
		return base_padding + num_columns * column_bytes; 
	}
}

/**
 * @brief Translates a dtype string and returns its dtype enumeration and any 
 * extended dtype flags that are supported by cuIO. Often, this is a column
 * with the same underlying dtype the basic types, but with different parsing
 * interpretations.
 *
 * @param[in] dtype String containing the basic or extended dtype
 *
 * @return std::pair<gdf_dtype, column_parse::flags> Tuple of dtype and flags
 */
std::pair<gdf_dtype, column_parse::flags> get_dtype_info(
    const std::string &dtype) {
  if (dtype == "hex" || dtype == "hex64") {
    return std::make_pair(GDF_INT64, column_parse::as_hexadecimal);
  }
  if (dtype == "hex32") {
    return std::make_pair(GDF_INT32, column_parse::as_hexadecimal);
  }

  return std::make_pair(convertStringToDtype(dtype), column_parse::as_default);
}

/**
* @brief Removes the first and Last quote in the string
*/
string removeQuotes(string str, char quotechar) {
	// Exclude first and last quotation char
	const size_t first_quote = str.find(quotechar);
	if (first_quote != string::npos) {
		str.erase(first_quote, 1);
	}
	const size_t  last_quote = str.rfind(quotechar);
	if (last_quote != string::npos) {
		str.erase(last_quote, 1);
	}

	return str;
}

/**
 * @brief Parse the first row to set the column names in the raw_csv parameter 
 *
 * The first row can be either the header row, or the first data row
 *
 * @return void
*/
void reader::Impl::setColumnNamesFromCsv() {
	vector<char> first_row = header;
	// No header, read the first data row
	if (first_row.empty()) {
		uint64_t first_row_len{};
		// If file only contains one row, recStart[1] is not valid
		if (num_records > 1) {
			CUDA_TRY(hipMemcpy(&first_row_len, recStart.data() + 1, sizeof(uint64_t), hipMemcpyDefault));
		}
		else {
			// File has one row - use the file size for the row size
			first_row_len = num_bytes / sizeof(char);
		}
		first_row.resize(first_row_len);
		CUDA_TRY(hipMemcpy(first_row.data(), data.data(), first_row_len * sizeof(char), hipMemcpyDefault));
	}

	int num_cols = 0;

	bool quotation	= false;
	for (size_t pos = 0, prev = 0; pos < first_row.size(); ++pos) {
		// Flip the quotation flag if current character is a quotechar
		if(first_row[pos] == opts.quotechar) {
			quotation = !quotation;
		}
		// Check if end of a column/row
		else if (pos == first_row.size() - 1 ||
				 (!quotation && first_row[pos] == opts.terminator) ||
				 (!quotation && first_row[pos] == opts.delimiter)) {
			// This is the header, add the column name
			if (args_.header >= 0) {
				// Include the current character, in case the line is not terminated
				int col_name_len = pos - prev + 1;
				// Exclude the delimiter/terminator is present
				if (first_row[pos] == opts.delimiter || first_row[pos] == opts.terminator) {
					--col_name_len;
				}
				// Also exclude '\r' character at the end of the column name if it's part of the terminator
				if (col_name_len > 0 &&
					opts.terminator == '\n' &&
					first_row[pos] == '\n' &&
					first_row[pos - 1] == '\r') {
					--col_name_len;
				}

				const string new_col_name(first_row.data() + prev, col_name_len);
				col_names.push_back(removeQuotes(new_col_name, opts.quotechar));

				// Stop parsing when we hit the line terminator; relevant when there is a blank line following the header.
				// In this case, first_row includes multiple line terminators at the end, as the new recStart belongs
				// to a line that comes after the blank line(s)
				if (!quotation && first_row[pos] == opts.terminator){
					break;
				}
			}
			else {
				// This is the first data row, add the automatically generated name
				col_names.push_back(args_.prefix + std::to_string(num_cols));
			}
			num_cols++;

			// Skip adjacent delimiters if delim_whitespace is set
			while (opts.multi_delimiter &&
				   pos < first_row.size() &&
				   first_row[pos] == opts.delimiter && 
				   first_row[pos + 1] == opts.delimiter) {
				++pos;
			}
			prev = pos + 1;
		}
	}
}

/**---------------------------------------------------------------------------*
 * @brief Updates the object with the total number of rows and
 * quotation characters in the file
 *
 * Does not count the quotations if quotechar is set to '/0'.
 *
 * @param[in] h_data Pointer to the csv data in host memory
 * @param[in] h_size Size of the input data, in bytes
 *
 * @return void
 *---------------------------------------------------------------------------**/
void reader::Impl::countRecordsAndQuotes(const char *h_data, size_t h_size) {
	vector<char> chars_to_count{opts.terminator};
	if (opts.quotechar != '\0') {
		chars_to_count.push_back(opts.quotechar);
	}

	num_records = countAllFromSet(h_data, h_size, chars_to_count);

	// If not starting at an offset, add an extra row to account for the first row in the file
	if (byte_range_offset == 0) {
		++num_records;
	}
}

/**---------------------------------------------------------------------------*
 * @brief Updates the object with the offset of each row in the file
 * Also add positions of each quotation character in the file.
 *
 * Does not process the quotations if quotechar is set to '/0'.
 *
 * @param[in] h_data Pointer to the csv data in host memory
 * @param[in] h_size Size of the input data, in bytes
 *
 * @return void
 *---------------------------------------------------------------------------**/
void reader::Impl::setRecordStarts(const char *h_data, size_t h_size) {
	// Allocate space to hold the record starting points
	const bool last_line_terminated = (h_data[h_size - 1] == opts.terminator);
	// If the last line is not terminated, allocate space for the EOF entry (added later)
	const gdf_size_type record_start_count = num_records + (last_line_terminated ? 0 : 1);
	recStart = device_buffer<uint64_t>(record_start_count); 

	auto* find_result_ptr = recStart.data();
	if (byte_range_offset == 0) {
		find_result_ptr++;
		CUDA_TRY(hipMemsetAsync(recStart.data(), 0ull, sizeof(uint64_t)));
	}
	vector<char> chars_to_find{opts.terminator};
	if (opts.quotechar != '\0') {
		chars_to_find.push_back(opts.quotechar);
	}
	// Passing offset = 1 to return positions AFTER the found character
	findAllFromSet(h_data, h_size, chars_to_find, 1, find_result_ptr);

	// Previous call stores the record pinput_file.typeositions as encountered by all threads
	// Sort the record positions as subsequent processing may require filtering
	// certain rows or other processing on specific records
	thrust::sort(rmm::exec_policy()->on(0), recStart.data(), recStart.data() + num_records);

	// Currently, ignoring lineterminations within quotes is handled by recording
	// the records of both, and then filtering out the records that is a quotechar
	// or a linetermination within a quotechar pair. The future major refactoring
	// of reader and its kernels will probably use a different tactic.
	if (opts.quotechar != '\0') {
		vector<uint64_t> h_rec_starts(num_records);
		const size_t rec_start_size = sizeof(uint64_t) * (h_rec_starts.size());
		CUDA_TRY( hipMemcpy(h_rec_starts.data(), recStart.data(), rec_start_size, hipMemcpyDeviceToHost) );

		auto recCount = num_records;

		bool quotation = false;
		for (gdf_size_type i = 1; i < num_records; ++i) {
			if (h_data[h_rec_starts[i] - 1] == opts.quotechar) {
				quotation = !quotation;
				h_rec_starts[i] = num_bytes;
				recCount--;
			}
			else if (quotation) {
				h_rec_starts[i] = num_bytes;
				recCount--;
			}
		}

		CUDA_TRY( hipMemcpy(recStart.data(), h_rec_starts.data(), rec_start_size, hipMemcpyHostToDevice) );
		thrust::sort(rmm::exec_policy()->on(0), recStart.data(), recStart.data() + num_records);
		num_records = recCount;
	}

	if (!last_line_terminated){
		// Add the EOF as the last record when the terminator is missing in the last line
		const uint64_t eof_offset = h_size;
		CUDA_TRY(hipMemcpy(recStart.data() + num_records, &eof_offset, sizeof(uint64_t), hipMemcpyDefault));
		// Update the record count
		++num_records;
	}
}

/**---------------------------------------------------------------------------*
 * @brief Reads CSV-structured data and returns an array of gdf_columns.
 *
 * @return void
 *---------------------------------------------------------------------------**/
table reader::Impl::read()
{
	// TODO move initialization to constructor
	num_actual_cols = args_.names.size();
	num_active_cols = args_.names.size();

	if (args_.delim_whitespace) {
		opts.delimiter = ' ';
		opts.multi_delimiter = true;
	} else {
		opts.delimiter = args_.delimiter;
		opts.multi_delimiter = false;
	}
	opts.terminator = args_.lineterminator;
	if (args_.quotechar != '\0' && args_.quoting != QUOTE_NONE) {
		opts.quotechar = args_.quotechar;
		opts.keepquotes = false;
		opts.doublequote = args_.doublequote;
	} else {
		opts.quotechar = '\0';
		opts.keepquotes = true;
		opts.doublequote = false;
	}
	opts.skipblanklines = args_.skip_blank_lines;
	opts.comment = args_.comment;
	opts.dayfirst = args_.dayfirst;
	opts.decimal = args_.decimal;
	opts.thousands = args_.thousands;
	CUDF_EXPECTS(opts.decimal != opts.delimiter, "Decimal point cannot be the same as the delimiter");
	CUDF_EXPECTS(opts.thousands != opts.delimiter, "Thousands separator cannot be the same as the delimiter");

  const auto compression_type = inferCompressionType(
      args_.compression, args_.input_data_form, args_.filepath_or_buffer,
      {{"gz", "gzip"}, {"zip", "zip"}, {"bz2", "bz2"}, {"xz", "xz"}});

  if (byte_range_offset > 0 || byte_range_size > 0) {
    CUDF_EXPECTS(compression_type == "none",
                 "Compression unsupported when reading using byte range");
  }

	// Handle user-defined booleans values, whereby field data is substituted
	// with true/false values; CUDF booleans are int types of 0 or 1
	vector<string> true_values{"True", "TRUE", "true"};
	true_values.insert(true_values.end(), args_.true_values.begin(), args_.true_values.end());

	d_trueTrie = createSerializedTrie(true_values);
	opts.trueValuesTrie = d_trueTrie.data().get();

	vector<string> false_values{"False", "FALSE", "false"};
	false_values.insert(false_values.end(), args_.false_values.begin(), args_.false_values.end());
	d_falseTrie = createSerializedTrie(false_values);
	opts.falseValuesTrie = d_falseTrie.data().get();

	if (args_.na_filter && (args_.keep_default_na || !args_.na_values.empty())) {
		vector<string> na_values{
			"#N/A", "#N/A N/A", "#NA", "-1.#IND", 
			"-1.#QNAN", "-NaN", "-nan", "1.#IND", 
			"1.#QNAN", "N/A", "NA", "NULL", 
			"NaN", "n/a", "nan", "null"};
		if(!args_.keep_default_na){
			na_values.clear();
		}
		na_values.insert(na_values.end(), args_.na_values.begin(), args_.na_values.end());

		d_naTrie = createSerializedTrie(na_values);
		opts.naValuesTrie = d_naTrie.data().get();
	}

  size_t range_size = 0;
  if (byte_range_size != 0) {
    const auto num_columns = std::max(args_.names.size(), args_.dtype.size());
    range_size = byte_range_size + calculateMaxRowSize(num_columns);
  }

  auto source = [&] {
    if (args_.input_data_form == FILE_PATH) {
      return datasource::create(args_.filepath_or_buffer, byte_range_offset,
                                range_size);
    } else if (args_.input_data_form == HOST_BUFFER) {
      return datasource::create(args_.filepath_or_buffer.c_str(),
                                args_.filepath_or_buffer.size());
    } else {
      CUDF_FAIL("Invalid input type");
    }
  }();

  // Return an empty dataframe if no data and no column metadata to process
  if (source->empty() && (args_.names.empty() || args_.dtype.empty())) {
    return table();
  }

  // Transfer source data to GPU
  if (not source->empty()) {
    const char *h_uncomp_data = nullptr;
    size_t h_uncomp_size = 0;

    num_bytes = (range_size != 0) ? range_size : source->size();
    const auto buffer = source->get_buffer(byte_range_offset, num_bytes);

    std::vector<char> h_uncomp_data_owner;
    if (compression_type == "none") {
      // Do not use the owner vector here to avoid extra copy
      h_uncomp_data = reinterpret_cast<const char *>(buffer->data());
      h_uncomp_size = buffer->size();
    } else {
      CUDF_EXPECTS(
          getUncompressedHostData(
              reinterpret_cast<const char *>(buffer->data()), buffer->size(),
              compression_type, h_uncomp_data_owner) == GDF_SUCCESS,
          "Cannot decompress data");
      h_uncomp_data = h_uncomp_data_owner.data();
      h_uncomp_size = h_uncomp_data_owner.size();
    }

    countRecordsAndQuotes(h_uncomp_data, h_uncomp_size);
    setRecordStarts(h_uncomp_data, h_uncomp_size);
    uploadDataToDevice(h_uncomp_data, h_uncomp_size);
  }

	//-----------------------------------------------------------------------------
	//-- Populate the header

  // Check if the user gave us a list of column names
  if (not args_.names.empty()) {
    h_column_flags.resize(args_.names.size(), column_parse::enabled);
    col_names = args_.names;
  } else {
    setColumnNamesFromCsv();

    num_actual_cols = num_active_cols = col_names.size();

    h_column_flags.resize(num_actual_cols, column_parse::enabled);

    // Rename empty column names to "Unnamed: col_index"
    for (size_t col_idx = 0; col_idx < col_names.size(); ++col_idx) {
      if (col_names[col_idx].empty()) {
        col_names[col_idx] = string("Unnamed: ") + std::to_string(col_idx);
      }
    }

    // Looking for duplicates
    std::unordered_map<string, int> col_names_histogram;
    for (auto& col_name: col_names){
      // Operator [] inserts a default-initialized value if the given key is not present
      if (++col_names_histogram[col_name] > 1){
        if (args_.mangle_dupe_cols) {
          // Rename duplicates of column X as X.1, X.2, ...; First appearance stays as X
          col_name += "." + std::to_string(col_names_histogram[col_name] - 1);
        }
        else {
          // All duplicate columns will be ignored; First appearance is parsed
          const auto idx = &col_name - col_names.data();
          h_column_flags[idx] = column_parse::disabled;
        }
      }
    }

    // Update the number of columns to be processed, if some might have been removed
    if (!args_.mangle_dupe_cols) {
      num_active_cols = col_names_histogram.size();
    }
  }

  // User can specify which columns should be parsed
  if (not args_.use_cols_indexes.empty() || not args_.use_cols_names.empty()) {
    std::fill(h_column_flags.begin(), h_column_flags.end(), column_parse::disabled);

    for (const auto index : args_.use_cols_indexes) {
      h_column_flags[index] = column_parse::enabled;
    }
    num_active_cols = args_.use_cols_indexes.size();

    for (const auto name : args_.use_cols_names) {
      const auto it = std::find(col_names.begin(), col_names.end(), name);
      if (it != col_names.end()) {
        h_column_flags[it - col_names.begin()] = column_parse::enabled;
        num_active_cols++;
      }
    }
  }

  // User can specify which columns should be inferred as datetime
  if (not args_.infer_date_indexes.empty() || not args_.infer_date_names.empty()) {
    for (auto index : args_.infer_date_indexes) {
      h_column_flags[index] |= column_parse::as_datetime;
    }
    for (auto name : args_.infer_date_names) {
      auto it = std::find(col_names.begin(), col_names.end(), name);
      if (it != col_names.end()) {
        h_column_flags[it - col_names.begin()] |= column_parse::as_datetime;
      }
    }
  }

	//-----------------------------------------------------------------------------
	//--- Auto detect types of the vectors

  if(args_.dtype.empty()){
    if (num_records == 0) {
      dtypes = vector<gdf_dtype>(num_active_cols, GDF_STRING);
    } else {
      vector<column_data_t> h_ColumnData(num_active_cols);
      device_buffer<column_data_t> d_ColumnData(num_active_cols);
      CUDA_TRY(hipMemset(d_ColumnData.data(), 0, sizeof(column_data_t) * num_active_cols));
      d_column_flags = h_column_flags;

      launch_dataTypeDetection(d_ColumnData.data());
      CUDA_TRY(hipMemcpy(h_ColumnData.data(), d_ColumnData.data(), sizeof(column_data_t) * num_active_cols, hipMemcpyDeviceToHost));

      // host: array of dtypes (since gdf_columns are not created until end)
      vector<gdf_dtype> d_detectedTypes;

      for(int col = 0; col < num_active_cols; col++){
        unsigned long long countInt = h_ColumnData[col].countInt8 + h_ColumnData[col].countInt16 +
                                      h_ColumnData[col].countInt32 + h_ColumnData[col].countInt64;

        if (h_ColumnData[col].countNULL == num_records){
          // Entire column is NULL; allocate the smallest amount of memory
          d_detectedTypes.push_back(GDF_INT8);
        } else if(h_ColumnData[col].countString > 0L){
          d_detectedTypes.push_back(GDF_STRING);
        } else if(h_ColumnData[col].countDateAndTime > 0L){
          d_detectedTypes.push_back(GDF_DATE64);
        } else if(h_ColumnData[col].countBool > 0L) {
          d_detectedTypes.push_back(GDF_BOOL8);
        } else if(h_ColumnData[col].countFloat > 0L ||
          (h_ColumnData[col].countFloat == 0L &&
           countInt > 0L && h_ColumnData[col].countNULL > 0L)) {
          // The second condition has been added to conform to
          // PANDAS which states that a column of integers with
          // a single NULL record need to be treated as floats.
          d_detectedTypes.push_back(GDF_FLOAT64);
        } else {
          // All other integers are stored as 64-bit to conform to PANDAS
          d_detectedTypes.push_back(GDF_INT64);
        }
      }
      dtypes = d_detectedTypes;
    }
  }
  else {
    const bool is_dict = std::all_of(
        args_.dtype.begin(), args_.dtype.end(),
        [](const auto &s) { return s.find(':') != std::string::npos; });

    if (!is_dict) {
      if (args_.dtype.size() == 1) {
        // If it's a single dtype, assign that dtype to all active columns
        const auto dtype_info = get_dtype_info(args_.dtype[0]);
        dtypes.resize(num_active_cols, dtype_info.first);
        for (int col = 0; col < num_actual_cols; col++) {
          h_column_flags[col] |= dtype_info.second;
        }
        CUDF_EXPECTS(dtypes.back() != GDF_invalid, "Unsupported data type");
      } else {
        // If it's a list, assign dtypes to active columns in the given order
        CUDF_EXPECTS(static_cast<int>(args_.dtype.size()) >= num_actual_cols,
                     "Must specify data types for all columns");
        for (int col = 0; col < num_actual_cols; col++) {
          if (h_column_flags[col] & column_parse::enabled) {
            const auto dtype_info = get_dtype_info(args_.dtype[col]);
            dtypes.push_back(dtype_info.first);
            h_column_flags[col] |= dtype_info.second;
            CUDF_EXPECTS(dtypes.back() != GDF_invalid, "Unsupported data type");
          }
        }
      }
    } else {
      // Translate vector of `name : dtype` strings to map
      // NOTE: Incoming pairs can be out-of-order from column names in dataset
      std::unordered_map<std::string, std::string> col_type_map;
      for (const auto& pair : args_.dtype) {
        const auto pos = pair.find_last_of(':');
        const auto name = pair.substr(0, pos);
        const auto dtype = pair.substr(pos + 1, pair.size());
        col_type_map[name] = dtype;
      }

      for (int col = 0; col < num_actual_cols; col++) {
        if (h_column_flags[col] & column_parse::enabled) {
          CUDF_EXPECTS(col_type_map.find(col_names[col]) != col_type_map.end(),
                       "Must specify data types for all active columns");
          const auto dtype_info = get_dtype_info(col_type_map[col_names[col]]);
          dtypes.push_back(dtype_info.first);
          h_column_flags[col] |= dtype_info.second;
          CUDF_EXPECTS(dtypes.back() != GDF_invalid, "Unsupported data type");
        }
      }
    }
  }
  // Alloc output; columns' data memory is still expected for empty dataframe
  std::vector<gdf_column_wrapper> columns;
  for (int col = 0, active_col = 0; col < num_actual_cols; ++col) {
    if (h_column_flags[col] & column_parse::enabled) {
      columns.emplace_back(num_records, dtypes[active_col],
                           gdf_dtype_extra_info{TIME_UNIT_NONE},
                           col_names[col]);
      CUDF_EXPECTS(columns.back().allocate() == GDF_SUCCESS, "Cannot allocate columns");
      active_col++;
    }
  }

  // Convert CSV input to cuDF output
  if (num_records != 0) {
    thrust::host_vector<gdf_dtype> h_dtypes(num_active_cols);
    thrust::host_vector<void*> h_data(num_active_cols);
    thrust::host_vector<gdf_valid_type*> h_valid(num_active_cols);

    for (int i = 0; i < num_active_cols; ++i) {
      h_dtypes[i] = columns[i]->dtype;
      h_data[i] = columns[i]->data;
      h_valid[i] = columns[i]->valid;
    }

    rmm::device_vector<gdf_dtype> d_dtypes = h_dtypes;
    rmm::device_vector<void*> d_data = h_data;
    rmm::device_vector<gdf_valid_type*> d_valid = h_valid;
    rmm::device_vector<gdf_size_type> d_valid_counts(num_active_cols, 0);
    d_column_flags = h_column_flags;

    launch_dataConvertColumns(d_data.data().get(), d_valid.data().get(), d_dtypes.data().get(),
                              d_valid_counts.data().get());
    CUDA_TRY(hipStreamSynchronize(0));

    thrust::host_vector<gdf_size_type> h_valid_counts = d_valid_counts;
    for (int i = 0; i < num_active_cols; ++i) {
      columns[i]->null_count = columns[i]->size - h_valid_counts[i];
    }
  }

  for (int i = 0; i < num_active_cols; ++i) {
    if (columns[i]->dtype == GDF_STRING) {
      std::unique_ptr<NVStrings, decltype(&NVStrings::destroy)> str_data(
        NVStrings::create_from_index(static_cast<string_pair *>(columns[i]->data), columns[i]->size), 
        &NVStrings::destroy);
      RMM_TRY(RMM_FREE(columns[i]->data, 0));

      // PANDAS' default behavior of enabling doublequote for two consecutive
      // quotechars in quoted fields results in reduction to a single quotechar
      if ((opts.quotechar != '\0') &&
          (opts.doublequote == true)) {
        const std::string quotechar(1, opts.quotechar);
        const std::string doublequotechar(2, opts.quotechar);
        columns[i]->data = str_data->replace(doublequotechar.c_str(), quotechar.c_str());
      }
      else {
        columns[i]->data = str_data.release();
      }
    }
  }

  // Transfer ownership to raw pointer output arguments
  std::vector<gdf_column *> out_cols(num_active_cols);
  for (size_t i = 0; i < out_cols.size(); ++i) {
    out_cols[i] = columns[i].release();
  }

  return table(out_cols.data(), out_cols.size());
}


/**---------------------------------------------------------------------------*
 * @brief Uploads the relevant segment of the input csv data onto the GPU.
 * 
 * Only rows that need to be read are copied to the GPU, based on parameters
 * like nrows, skipheader, skipfooter.
 * Also updates the array of record starts to match the device data offset.
 * 
 * @param[in] h_uncomp_data Pointer to the uncompressed csv data in host memory
 * @param[in] h_uncomp_size Size of the input data, in bytes
 * 
 * @return void
 *---------------------------------------------------------------------------**/
void reader::Impl::uploadDataToDevice(const char *h_uncomp_data, size_t h_uncomp_size) {

  // Exclude the rows that are to be skipped from the start
  CUDF_EXPECTS(num_records > skiprows, "Skipping too many rows");
  const auto first_row = skiprows;
  num_records = num_records - first_row;

  std::vector<uint64_t> h_rec_starts(num_records);
  CUDA_TRY(hipMemcpy(h_rec_starts.data(), recStart.data() + first_row,
                      sizeof(uint64_t) * h_rec_starts.size(),
                      hipMemcpyDefault));

  // Trim lines that are outside range, but keep one greater for the end offset
  if (byte_range_size != 0) {
    auto it = h_rec_starts.end() - 1;
    while (it >= h_rec_starts.begin() &&
           *it > uint64_t(byte_range_size)) {
      --it;
    }
    if ((it + 2) < h_rec_starts.end()) {
      h_rec_starts.erase(it + 2, h_rec_starts.end());
    }
  }

  // Discard only blank lines, only fully comment lines, or both.
  // If only handling one of them, ensure it doesn't match against \0 as we do
  // not want certain scenarios to be filtered out (end-of-file)
  if (opts.skipblanklines || opts.comment != '\0') {
    const auto match_newline = opts.skipblanklines ? opts.terminator
                                                            : opts.comment;
    const auto match_comment = opts.comment != '\0' ? opts.comment
                                                             : match_newline;
    const auto match_return = (opts.skipblanklines &&
                              opts.terminator == '\n') ? '\r'
                                                                : match_comment;
    h_rec_starts.erase(
        std::remove_if(h_rec_starts.begin(), h_rec_starts.end(),
                       [&](uint64_t i) {
                         return (h_uncomp_data[i] == match_newline ||
                                 h_uncomp_data[i] == match_return ||
                                 h_uncomp_data[i] == match_comment);
                       }),
        h_rec_starts.end());
  }

  num_records = h_rec_starts.size();

  // Exclude the rows before the header row (inclusive)
  // But copy the header data for parsing the column names later (if necessary)
  if (args_.header >= 0) {
    header.assign(
        h_uncomp_data + h_rec_starts[args_.header],
        h_uncomp_data + h_rec_starts[args_.header + 1]);
    h_rec_starts.erase(h_rec_starts.begin(),
                       h_rec_starts.begin() + args_.header + 1);
    num_records = h_rec_starts.size();
  }

  // Exclude the rows that exceed past the requested number
  if (nrows >= 0 && nrows < num_records) {
    h_rec_starts.resize(nrows + 1);    // include end offset
    num_records = h_rec_starts.size();
  }

  // Exclude the rows that are to be skipped from the end
  if (skipfooter > 0) {
    h_rec_starts.resize(h_rec_starts.size() - skipfooter);
    num_records = h_rec_starts.size();
  }

  CUDF_EXPECTS(num_records > 0, "No data available for parsing");

  const auto start_offset = h_rec_starts.front();
  const auto end_offset = h_rec_starts.back();
  num_bytes = end_offset - start_offset;
  assert(num_bytes <= h_uncomp_size);
  num_bits = (num_bytes + 63) / 64;

  // Resize and upload the rows of interest
  recStart.resize(num_records);
  CUDA_TRY(hipMemcpy(recStart.data(), h_rec_starts.data(),
                      sizeof(uint64_t) * num_records,
                      hipMemcpyDefault));

  // Upload the raw data that is within the rows of interest
  data = device_buffer<char>(num_bytes);
  CUDA_TRY(hipMemcpy(data.data(), h_uncomp_data + start_offset,
                      num_bytes, hipMemcpyHostToDevice));

  // Adjust row start positions to account for the data subcopy
  thrust::transform(rmm::exec_policy()->on(0), recStart.data(),
                    recStart.data() + num_records,
                    thrust::make_constant_iterator(start_offset),
                    recStart.data(), thrust::minus<uint64_t>());

  // The array of row offsets includes EOF
  // reduce the number of records by one to exclude it from the row count
  num_records--;
}

/**---------------------------------------------------------------------------*
 * @brief Helper function to setup and launch CSV parsing CUDA kernel.
 * 
 * @param[out] gdf The output column data
 * @param[out] valid The bitmaps indicating whether column fields are valid
 * @param[in] d_dtypes The data types of the columns
 * @param[out] num_valid The numbers of valid fields in columns
 *---------------------------------------------------------------------------**/
void reader::Impl::launch_dataConvertColumns(void **gdf, gdf_valid_type **valid,
                                             gdf_dtype *d_dtypes,
                                             gdf_size_type *num_valid) {
  int blockSize;    // suggested thread count to use
  int minGridSize;  // minimum block count required
  CUDA_TRY(hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize,
                                              convertCsvToGdf));

  // Calculate actual block count to use based on records count
  const int gridSize = (num_records + blockSize - 1) / blockSize;

  convertCsvToGdf <<< gridSize, blockSize >>> (
      data.data(), opts, num_records, num_actual_cols,
      d_column_flags.data().get(), recStart.data(), d_dtypes, gdf, valid,
      num_valid);

  CUDA_TRY(hipGetLastError());
}

/**---------------------------------------------------------------------------*
 * @brief Functor for converting CSV data to cuDF data type value.
 *---------------------------------------------------------------------------**/
struct ConvertFunctor {
  /**---------------------------------------------------------------------------*
   * @brief Template specialization for operator() for types whose values can be
   * convertible to a 0 or 1 to represent false/true. The converting is done by
   * checking against the default and user-specified true/false values list.
   *
   * It is handled here rather than within convertStrToValue() as that function
   * is used by other types (ex. timestamp) that aren't 'booleable'.
   *---------------------------------------------------------------------------**/
  template <typename T,
            typename std::enable_if_t<std::is_integral<T>::value> * = nullptr>
  __host__ __device__ __forceinline__ void operator()(
      const char *csvData, void *gdfColumnData, long rowIndex, long start,
      long end, const ParseOptions &opts, column_parse::flags flags) {
    T &value{static_cast<T *>(gdfColumnData)[rowIndex]};

    // Check for user-specified true/false values first, where the output is
    // replaced with 1/0 respectively
    const size_t field_len = end - start + 1;
    if (serializedTrieContains(opts.trueValuesTrie, csvData + start, field_len)) {
      value = 1;
    } else if (serializedTrieContains(opts.falseValuesTrie, csvData + start, field_len)) {
      value = 0;
    } else {
      if (flags & column_parse::as_hexadecimal) {
        value = convertStrToValue<T, 16>(csvData, start, end, opts);
      } else {
        value = convertStrToValue<T>(csvData, start, end, opts);
      }
    }
  }

  /**---------------------------------------------------------------------------*
   * @brief Default template operator() dispatch specialization all data types
   * (including wrapper types) that is not covered by above.
   *---------------------------------------------------------------------------**/
  template <typename T,
            typename std::enable_if_t<!std::is_integral<T>::value> * = nullptr>
  __host__ __device__ __forceinline__ void operator()(
      const char *csvData, void *gdfColumnData, long rowIndex, long start,
      long end, const ParseOptions &opts, column_parse::flags flags) {
    T &value{static_cast<T *>(gdfColumnData)[rowIndex]};
    value = convertStrToValue<T>(csvData, start, end, opts);
  }
};

/**---------------------------------------------------------------------------*
 * @brief CUDA kernel that parses and converts CSV data into cuDF column data.
 * 
 * Data is processed one record at a time
 *
 * @param[in] raw_csv The entire CSV data to read
 * @param[in] opts A set of parsing options
 * @param[in] num_records The number of lines/rows of CSV data
 * @param[in] num_columns The number of columns of CSV data
 * @param[in] column_flags Per-column parsing behavior flags
 * @param[in] recStart The start the CSV data of interest
 * @param[in] dtype The data type of the column
 * @param[out] data The output column data
 * @param[out] valid The bitmaps indicating whether column fields are valid
 * @param[out] num_valid The numbers of valid fields in columns
 *---------------------------------------------------------------------------**/
__global__ void convertCsvToGdf(char *raw_csv, const ParseOptions opts,
                                gdf_size_type num_records, int num_columns,
                                column_parse::flags *flags, uint64_t *recStart,
                                gdf_dtype *dtype, void **data,
                                gdf_valid_type **valid,
                                gdf_size_type *num_valid) {

	// thread IDs range per block, so also need the block id
	long	rec_id  = threadIdx.x + (blockDim.x * blockIdx.x);		// this is entry into the field array - tid is an elements within the num_entries array

	// we can have more threads than data, make sure we are not past the end of the data
	if ( rec_id >= num_records)
		return;

	long start 		= recStart[rec_id];
	long stop 		= recStart[rec_id + 1];

	long pos 		= start;
	int  col 		= 0;
	int  actual_col = 0;

	while(col<num_columns){

		if(start>stop)
			break;

		pos = seekFieldEnd(raw_csv, opts, pos, stop);

		if (flags[col] & column_parse::enabled) {

			// check if the entire field is a NaN string - consistent with pandas
			const bool is_na = serializedTrieContains(opts.naValuesTrie, raw_csv + start, pos - start);

			// Modify start & end to ignore whitespace and quotechars
			long tempPos=pos-1;
			if(!is_na && dtype[actual_col] != gdf_dtype::GDF_CATEGORY && dtype[actual_col] != gdf_dtype::GDF_STRING){
				adjustForWhitespaceAndQuotes(raw_csv, &start, &tempPos, opts.quotechar);
			}

			if(!is_na && start<=(tempPos)) { // Empty fields are not legal values

				// Type dispatcher does not handle GDF_STRINGS
				if (dtype[actual_col] == gdf_dtype::GDF_STRING) {
					long end = pos;
					if(opts.keepquotes==false){
						if((raw_csv[start] == opts.quotechar) && (raw_csv[end-1] == opts.quotechar)){
							start++;
							end--;
						}
					}
					auto str_list = static_cast<string_pair*>(data[actual_col]);
					str_list[rec_id].first = raw_csv + start;
					str_list[rec_id].second = end - start;
				} else {
					cudf::type_dispatcher(
						dtype[actual_col], ConvertFunctor{}, raw_csv,
						data[actual_col], rec_id, start, tempPos, opts, flags[col]);
				}

				// set the valid bitmap - all bits were set to 0 to start
				setBitmapBit(valid[actual_col], rec_id);
				atomicAdd(&num_valid[actual_col], 1);
			}
			else if(dtype[actual_col]==gdf_dtype::GDF_STRING){
				auto str_list = static_cast<string_pair*>(data[actual_col]);
				str_list[rec_id].first = nullptr;
				str_list[rec_id].second = 0;
			}
			actual_col++;
		}
		pos++;
		start=pos;
		col++;

	}
}

/**---------------------------------------------------------------------------*
 * @brief Helper function to setup and launch CSV data type detect CUDA kernel.
 * 
 * @param[in] raw_csv The metadata for the CSV data
 * @param[out] d_columnData The count for each column data type
 *
 * @return void
 *---------------------------------------------------------------------------**/
 void reader::Impl::launch_dataTypeDetection(column_data_t *d_columnData) {
  int blockSize;    // suggested thread count to use
  int minGridSize;  // minimum block count required
  CUDA_TRY(hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize,
                                              dataTypeDetection));

  // Calculate actual block count to use based on records count
  const int gridSize = (num_records + blockSize - 1) / blockSize;

  dataTypeDetection <<< gridSize, blockSize >>> (
      data.data(), opts, num_records, num_actual_cols,
      d_column_flags.data().get(), recStart.data(), d_columnData);

  CUDA_TRY(hipGetLastError());
}

/**---------------------------------------------------------------------------*
 * @brief CUDA kernel that parses and converts CSV data into cuDF column data.
 *
 * Data is processed in one row/record at a time, so the number of total
 * threads (tid) is equal to the number of rows.
 *
 * @param[in] raw_csv The entire CSV data to read
 * @param[in] opts A set of parsing options
 * @param[in] num_records The number of lines/rows of CSV data
 * @param[in] num_columns The number of columns of CSV data
 * @param[in] column_flags Per-column parsing behavior flags
 * @param[in] recStart The start the CSV data of interest
 * @param[out] d_columnData The count for each column data type
 *---------------------------------------------------------------------------**/
__global__ void dataTypeDetection(char *raw_csv, const ParseOptions opts,
                                  gdf_size_type num_records, int num_columns,
                                  column_parse::flags *flags,
                                  uint64_t *recStart,
                                  column_data_t *d_columnData) {

  // ThreadIds range per block, so also need the blockId
  // This is entry into the fields; threadId is an element within `num_records`
  long rec_id = threadIdx.x + (blockDim.x * blockIdx.x);

  // we can have more threads than data, make sure we are not past the end of the data
  if (rec_id >= num_records) {
    return;
  }

  long start = recStart[rec_id];
  long stop = recStart[rec_id + 1];

  long pos = start;
  int col = 0;
  int actual_col = 0;

  // Going through all the columns of a given record
  while (col < num_columns) {
    if (start > stop) {
      break;
    }

    pos = seekFieldEnd(raw_csv, opts, pos, stop);

    // Checking if this is a column that the user wants --- user can filter columns
    if (flags[col] & column_parse::enabled) {
      long tempPos = pos - 1;
      long field_len = pos - start;

      if (field_len <= 0 ||
          serializedTrieContains(opts.naValuesTrie, raw_csv + start,
                                 field_len)) {
        atomicAdd(&d_columnData[actual_col].countNULL, 1);
      } else if (serializedTrieContains(opts.trueValuesTrie, raw_csv + start,
                                        field_len) ||
                 serializedTrieContains(opts.falseValuesTrie, raw_csv + start,
                                        field_len)) {
        atomicAdd(&d_columnData[actual_col].countBool, 1);
      } else {
        long countNumber = 0;
        long countDecimal = 0;
        long countSlash = 0;
        long countDash = 0;
        long countPlus = 0;
        long countColon = 0;
        long countString = 0;
        long countExponent = 0;

        // Modify start & end to ignore whitespace and quotechars
        // This could possibly result in additional empty fields
        adjustForWhitespaceAndQuotes(raw_csv, &start, &tempPos);
        field_len = tempPos - start + 1;

        for (long startPos = start; startPos <= tempPos; startPos++) {
          if (isDigit(raw_csv[startPos])) {
            countNumber++;
            continue;
          }
          // Looking for unique characters that will help identify column types.
          switch (raw_csv[startPos]) {
            case '.':
              countDecimal++;
              break;
            case '-':
              countDash++;
              break;
            case '+':
              countPlus++;
              break;
            case '/':
              countSlash++;
              break;
            case ':':
              countColon++;
              break;
            case 'e':
            case 'E':
              if (startPos > start && startPos < tempPos)
                countExponent++;
              break;
            default:
              countString++;
              break;
          }
        }

        // Integers have to have the length of the string
        long int_req_number_cnt = field_len;
        // Off by one if they start with a minus sign
        if ((raw_csv[start] == '-' || raw_csv[start] == '+') && field_len > 1) {
          --int_req_number_cnt;
        }

        if (field_len == 0) {
          // Ignoring whitespace and quotes can result in empty fields
          atomicAdd(&d_columnData[actual_col].countNULL, 1);
        } else if (flags[col] & column_parse::as_datetime) {
          // PANDAS uses `object` dtype if the date is unparseable
          if (isLikeDateTime(countString, countDecimal, countColon, countDash,
                             countSlash)) {
            atomicAdd(&d_columnData[actual_col].countDateAndTime, 1);
          } else {
            atomicAdd(&d_columnData[actual_col].countString, 1);
          }
        } else if (countNumber == int_req_number_cnt) {
          // Checking to see if we the integer value requires 8,16,32,64 bits.
          // This will allow us to allocate the exact amount of memory.
          const auto value =
              convertStrToValue<int64_t>(raw_csv, start, tempPos, opts);
          if (value >= (1L << 31)) {
            atomicAdd(&d_columnData[actual_col].countInt64, 1);
          } else if (value >= (1L << 15)) {
            atomicAdd(&d_columnData[actual_col].countInt32, 1);
          } else if (value >= (1L << 7)) {
            atomicAdd(&d_columnData[actual_col].countInt16, 1);
          } else {
            atomicAdd(&d_columnData[actual_col].countInt8, 1);
          }
        } else if (isLikeFloat(field_len, countNumber, countDecimal,
                               countDash + countPlus, countExponent)) {
          atomicAdd(&d_columnData[actual_col].countFloat, 1);
        } else {
          atomicAdd(&d_columnData[actual_col].countString, 1);
        }
      }
      actual_col++;
    }
    pos++;
    start = pos;
    col++;
  }
}

reader::Impl::Impl(reader_options const &args) : args_(args) {}

table reader::Impl::read_byte_range(size_t offset, size_t size) {
  byte_range_offset = offset;
  byte_range_size = size;
  return read();
}

table reader::Impl::read_rows(gdf_size_type num_skip_header,
                              gdf_size_type num_skip_footer,
                              gdf_size_type num_rows) {
  CUDF_EXPECTS(num_rows == -1 || num_skip_footer == 0,
               "cannot use both num_rows and num_skip_footer parameters");

  skiprows = num_skip_header;
  nrows = num_rows;
  skipfooter = num_skip_footer;
  return read();
}

reader::reader(reader_options const &args)
    : impl_(std::make_unique<Impl>(args)) {}

table reader::read() { return impl_->read(); }

table reader::read_byte_range(size_t offset, size_t size) {
  return impl_->read_byte_range(offset, size);
}
table reader::read_rows(gdf_size_type num_skip_header,
                        gdf_size_type num_skip_footer, gdf_size_type num_rows) {
  return impl_->read_rows(num_skip_header, num_skip_footer, num_rows);
}

reader::~reader() = default;

} // namespace csv
} // namespace io
} // namespace cudf
