#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2018, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
	 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

/**
 * @file csv-reader.cu  code to read csv data
 *
 * CSV Reader
 */

#include "csv_gpu.h"
#include "csv_reader_impl.hpp"

#include <hip/hip_runtime.h>

#include <algorithm>
#include <iostream>
#include <numeric>
#include <string>
#include <vector>
#include <tuple>
#include <utility>
#include <iterator>
#include <memory>
#include <unordered_map>
#include <cstring>

#include <stdio.h>
#include <stdlib.h>

#include <unistd.h>
#include <fcntl.h>
#include <sys/types.h>
#include <sys/stat.h>
#include <sys/mman.h>

#include <thrust/device_ptr.h>
#include <thrust/execution_policy.h>
#include <thrust/host_vector.h>

#include "type_conversion.cuh"
#include "datetime_parser.cuh"

#include <cudf/cudf.h>
#include <cudf/unary.hpp>
#include <utilities/error_utils.hpp>
#include <utilities/trie.cuh>
#include <cudf/utilities/legacy/type_dispatcher.hpp>
#include <utilities/cudf_utils.h> 

#include <nvstrings/NVStrings.h>

#include <rmm/rmm.h>
#include <rmm/thrust_rmm_allocator.h>
#include <io/comp/io_uncomp.h>

#include <io/cuio_common.hpp>
#include <io/utilities/datasource.hpp>
#include <io/utilities/parsing_utils.cuh>

using std::vector;
using std::string;

namespace cudf {
namespace io {
namespace csv {


/**---------------------------------------------------------------------------*
 * @brief Estimates the maximum expected length or a row, based on the number 
 * of columns
 * 
 * If the number of columns is not available, it will return a value large 
 * enough for most use cases
 * 
 * @param[in] num_columns Number of columns in the CSV file (optional)
 * 
 * @return Estimated maximum size of a row, in bytes
 *---------------------------------------------------------------------------**/
 constexpr size_t calculateMaxRowSize(int num_columns=0) noexcept {
	constexpr size_t max_row_bytes = 16*1024; // 16KB
	constexpr size_t column_bytes = 64;
	constexpr size_t base_padding = 1024; // 1KB
	if (num_columns == 0){
		// Use flat size if the number of columns is not known
		return max_row_bytes;
	}
	else {
		// Expand the size based on the number of columns, if available
		return base_padding + num_columns * column_bytes; 
	}
}

/**
 * @brief Translates a dtype string and returns its dtype enumeration and any 
 * extended dtype flags that are supported by cuIO. Often, this is a column
 * with the same underlying dtype the basic types, but with different parsing
 * interpretations.
 *
 * @param[in] dtype String containing the basic or extended dtype
 *
 * @return std::pair<gdf_dtype, column_parse::flags> Tuple of dtype and flags
 */
std::tuple<gdf_dtype, gdf_dtype_extra_info, column_parse::flags> get_dtype_info(
    const std::string &dtype) {

  if (dtype == "hex" || dtype == "hex64") {
    return std::make_tuple(GDF_INT64, gdf_dtype_extra_info{ TIME_UNIT_NONE }, column_parse::as_hexadecimal);
  }
  if (dtype == "hex32") {
    return std::make_tuple(GDF_INT32, gdf_dtype_extra_info{ TIME_UNIT_NONE }, column_parse::as_hexadecimal);
  }

  gdf_dtype out_dtype;
  gdf_dtype_extra_info out_info{};
  std::tie(out_dtype, out_info) = convertStringToDtype(dtype);

  return std::make_tuple(out_dtype, out_info, column_parse::as_default);
}

/**
* @brief Removes the first and Last quote in the string
*/
string removeQuotes(string str, char quotechar) {
	// Exclude first and last quotation char
	const size_t first_quote = str.find(quotechar);
	if (first_quote != string::npos) {
		str.erase(first_quote, 1);
	}
	const size_t  last_quote = str.rfind(quotechar);
	if (last_quote != string::npos) {
		str.erase(last_quote, 1);
	}

	return str;
}

/**
 * @brief Parse the first row to set the column names in the raw_csv parameter 
 *
 * The first row can be either the header row, or the first data row
 *
 * @return void
*/
void reader::Impl::setColumnNamesFromCsv() {
  // If there is only a single character then it would be the terminator
  if (header.size() <= 1) {
    return;
  }

	std::vector<char> first_row = header;
	int num_cols = 0;

	bool quotation	= false;
	for (size_t pos = 0, prev = 0; pos < first_row.size(); ++pos) {
		// Flip the quotation flag if current character is a quotechar
		if(first_row[pos] == opts.quotechar) {
			quotation = !quotation;
		}
		// Check if end of a column/row
		else if (pos == first_row.size() - 1 ||
				 (!quotation && first_row[pos] == opts.terminator) ||
				 (!quotation && first_row[pos] == opts.delimiter)) {
			// This is the header, add the column name
			if (args_.header >= 0) {
				// Include the current character, in case the line is not terminated
				int col_name_len = pos - prev + 1;
				// Exclude the delimiter/terminator is present
				if (first_row[pos] == opts.delimiter || first_row[pos] == opts.terminator) {
					--col_name_len;
				}
				// Also exclude '\r' character at the end of the column name if it's part of the terminator
				if (col_name_len > 0 &&
					opts.terminator == '\n' &&
					first_row[pos] == '\n' &&
					first_row[pos - 1] == '\r') {
					--col_name_len;
				}

				const string new_col_name(first_row.data() + prev, col_name_len);
				col_names.push_back(removeQuotes(new_col_name, opts.quotechar));

				// Stop parsing when we hit the line terminator; relevant when there is a blank line following the header.
				// In this case, first_row includes multiple line terminators at the end, as the new recStart belongs
				// to a line that comes after the blank line(s)
				if (!quotation && first_row[pos] == opts.terminator){
					break;
				}
			}
			else {
				// This is the first data row, add the automatically generated name
				col_names.push_back(args_.prefix + std::to_string(num_cols));
			}
			num_cols++;

			// Skip adjacent delimiters if delim_whitespace is set
			while (opts.multi_delimiter &&
				   pos < first_row.size() &&
				   first_row[pos] == opts.delimiter && 
				   first_row[pos + 1] == opts.delimiter) {
				++pos;
			}
			prev = pos + 1;
		}
	}
}

table reader::Impl::read(size_t range_offset, size_t range_size,
                         gdf_size_type skip_rows, gdf_size_type skip_end_rows,
                         gdf_size_type num_rows) {
  if (range_offset > 0 || range_size > 0) {
    CUDF_EXPECTS(compression_type_ == "none",
                 "Reading compressed data using `byte range` is unsupported");
  }
  size_t map_range_size = 0;
  if (range_size != 0) {
    const auto num_columns = std::max(args_.names.size(), args_.dtype.size());
    map_range_size = range_size + calculateMaxRowSize(num_columns);
  }

  // Support delayed opening of the file if using memory mapping datasource
  // This allows only mapping of a subset of the file if using byte range
  if (source_ == nullptr) {
    assert(!filepath_.empty());
    source_ = datasource::create(filepath_, range_offset, map_range_size);
  }

  // Return an empty dataframe if no data and no column metadata to process
  if (source_->empty() && (args_.names.empty() || args_.dtype.empty())) {
    return cudf::table{};
  }

  // Transfer source data to GPU
  if (!source_->empty()) {
    const char *h_uncomp_data = nullptr;
    size_t h_uncomp_size = 0;

    auto data_size = (map_range_size != 0) ? map_range_size : source_->size();
    auto buffer = source_->get_buffer(range_offset, data_size);

    std::vector<char> h_uncomp_data_owner;
    if (compression_type_ == "none") {
      // Do not use the owner vector here to avoid extra copy
      h_uncomp_data = reinterpret_cast<const char *>(buffer->data());
      h_uncomp_size = buffer->size();
    } else {
      CUDF_EXPECTS(
          getUncompressedHostData(
              reinterpret_cast<const char *>(buffer->data()), buffer->size(),
              compression_type_, h_uncomp_data_owner) == GDF_SUCCESS,
          "Cannot decompress data");
      h_uncomp_data = h_uncomp_data_owner.data();
      h_uncomp_size = h_uncomp_data_owner.size();
    }

    gather_row_offsets(h_uncomp_data, h_uncomp_size, range_offset);
    auto row_range = select_rows(h_uncomp_data, h_uncomp_size, range_size,
                                 skip_rows, skip_end_rows, num_rows);

    data_size = row_range.second - row_range.first;
    CUDF_EXPECTS(data_size <= h_uncomp_size, "Row range exceeds data size");

    num_bits = (data_size + 63) / 64;
    data = device_buffer<char>(data_size);
    CUDA_TRY(hipMemcpyAsync(data.data(), h_uncomp_data + row_range.first,
                             data_size, hipMemcpyHostToDevice));
  }

  // Check if the user gave us a list of column names
  if (not args_.names.empty()) {
    h_column_flags.resize(args_.names.size(), column_parse::enabled);
    col_names = args_.names;
  } else {
    setColumnNamesFromCsv();

    num_actual_cols = num_active_cols = col_names.size();

    h_column_flags.resize(num_actual_cols, column_parse::enabled);

    // Rename empty column names to "Unnamed: col_index"
    for (size_t col_idx = 0; col_idx < col_names.size(); ++col_idx) {
      if (col_names[col_idx].empty()) {
        col_names[col_idx] = string("Unnamed: ") + std::to_string(col_idx);
      }
    }

    // Looking for duplicates
    std::unordered_map<string, int> col_names_histogram;
    for (auto& col_name: col_names){
      // Operator [] inserts a default-initialized value if the given key is not present
      if (++col_names_histogram[col_name] > 1){
        if (args_.mangle_dupe_cols) {
          // Rename duplicates of column X as X.1, X.2, ...; First appearance stays as X
          col_name += "." + std::to_string(col_names_histogram[col_name] - 1);
        }
        else {
          // All duplicate columns will be ignored; First appearance is parsed
          const auto idx = &col_name - col_names.data();
          h_column_flags[idx] = column_parse::disabled;
        }
      }
    }

    // Update the number of columns to be processed, if some might have been removed
    if (!args_.mangle_dupe_cols) {
      num_active_cols = col_names_histogram.size();
    }
  }

  // User can specify which columns should be parsed
  if (!args_.use_cols_indexes.empty() || !args_.use_cols_names.empty()) {
    std::fill(h_column_flags.begin(), h_column_flags.end(), column_parse::disabled);

    for (const auto index : args_.use_cols_indexes) {
      h_column_flags[index] = column_parse::enabled;
    }
    num_active_cols = args_.use_cols_indexes.size();

    for (const auto name : args_.use_cols_names) {
      const auto it = std::find(col_names.begin(), col_names.end(), name);
      if (it != col_names.end()) {
        h_column_flags[it - col_names.begin()] = column_parse::enabled;
        num_active_cols++;
      }
    }
  }

  // User can specify which columns should be inferred as datetime
  if (!args_.infer_date_indexes.empty() || !args_.infer_date_names.empty()) {
    for (const auto index : args_.infer_date_indexes) {
      h_column_flags[index] |= column_parse::as_datetime;
    }
    
    for (const auto name : args_.infer_date_names) {
      auto it = std::find(col_names.begin(), col_names.end(), name);
      if (it != col_names.end()) {
        h_column_flags[it - col_names.begin()] |= column_parse::as_datetime;
      }
    }
  }

  // Return empty table rather than exception if nothing to load
  if (num_active_cols == 0) {
    return cudf::table{};
  }

  std::vector<gdf_dtype> dtypes{};
  std::vector<gdf_dtype_extra_info> dtypes_extra_info{};
  std::tie(dtypes, dtypes_extra_info) = gather_column_dtypes();

  // Alloc output; columns' data memory is still expected for empty dataframe
  std::vector<gdf_column_wrapper> columns;
  for (int col = 0, active_col = 0; col < num_actual_cols; ++col) {
    if (h_column_flags[col] & column_parse::enabled) {
      columns.emplace_back(num_records, dtypes[active_col],
                           dtypes_extra_info[active_col],
                           col_names[col]);
      columns.back().allocate();
      active_col++;
    }
  }

  // Convert CSV input to cuDF output
  if (num_records != 0) {
    decode_data(columns);
  }

  // Perform any final column preparation (may reference decoded data)
  for (auto &column : columns) {
    column.finalize();

    // PANDAS' default behavior of enabling doublequote for two consecutive
    // quotechars in quoted fields results in reduction to a single quotechar
    if (column->dtype == GDF_STRING &&
        (opts.quotechar != '\0' && opts.doublequote == true)) {
      const std::string quotechar(1, opts.quotechar);
      const std::string dblquotechar(2, opts.quotechar);
      auto str_data = static_cast<NVStrings *>(column->data);
      column->data = str_data->replace(dblquotechar.c_str(), quotechar.c_str());
      NVStrings::destroy(str_data);
    }
  }

  // Transfer ownership to raw pointer output arguments
  std::vector<gdf_column *> out_cols(columns.size());

  auto maybe_cast_datetimes = args_.out_time_unit != TIME_UNIT_NONE;

  for (size_t i = 0; i < columns.size(); ++i) {
    if (maybe_cast_datetimes) {

      auto is_datetime = columns[i]->dtype == GDF_DATE32 ||
                         columns[i]->dtype == GDF_DATE64 ||
                         columns[i]->dtype == GDF_TIMESTAMP;

      if (is_datetime && columns[i]->dtype_info.time_unit != args_.out_time_unit) {
        // Cast the datetime-like column to the desired out_time_unit
        auto col = columns[i].get();
        gdf_dtype_extra_info dtype_info{args_.out_time_unit};
        auto res = cudf::cast(*col, GDF_TIMESTAMP, dtype_info);
        // Now free the original device memory
        gdf_column_free(col);
        // Assign the cast result to the output column
        col->size = res.size;
        col->data = res.data;
        col->valid = res.valid;
        col->dtype = res.dtype;
        col->dtype_info = res.dtype_info;
        col->null_count = res.null_count;
      }
    }
    out_cols[i] = columns[i].release();
  }

  return cudf::table(out_cols.data(), out_cols.size());
}

void reader::Impl::gather_row_offsets(const char *h_data, size_t h_size,
                                      size_t range_offset) {
  // Account for the start and end of row region offsets
  const bool require_first_line_start = (range_offset == 0);
  const bool require_last_line_end = (h_data[h_size - 1] != opts.terminator);

  auto symbols = (opts.quotechar != '\0')
                     ? std::vector<char>{opts.terminator, opts.quotechar}
                     : std::vector<char>{opts.terminator};
  const auto num_rows = countAllFromSet(h_data, h_size, symbols) +
                        (require_first_line_start ? 1 : 0);
  const auto num_offsets = num_rows + (require_last_line_end ? 1 : 0);
  row_offsets.resize(num_offsets);

  auto ptr_first = row_offsets.data().get();
  auto ptr_last = ptr_first + num_rows;
  if (require_first_line_start) {
    ptr_first++;
    const uint64_t first_entry = 0;
    row_offsets.front() = first_entry;
  }
  if (require_last_line_end) {
    const uint64_t last_entry = h_size;
    row_offsets.back() = last_entry;
  }

  // Passing offset = 1 to return positions AFTER the found character
  findAllFromSet(h_data, h_size, symbols, 1, ptr_first);

  // Sort the row info according to ascending start offset
  // Subsequent processing (filtering, etc.) may require row order
  thrust::sort(rmm::exec_policy()->on(0), ptr_first, ptr_last);
}

std::pair<uint64_t, uint64_t> reader::Impl::select_rows(
    const char *h_data, size_t h_size, size_t range_size,
    gdf_size_type skip_rows, gdf_size_type skip_end_rows,
    gdf_size_type num_rows) {
  thrust::host_vector<uint64_t> h_row_offsets = row_offsets;
  auto it_begin = h_row_offsets.begin();
  auto it_end = h_row_offsets.end();
  assert(std::distance(it_begin, it_end) >= 1);

  // Currently, ignoring lineterminations within quotes is handled by recording
  // the records of both, and then filtering out the records that is a quotechar
  // or a linetermination within a quotechar pair.
  if (opts.quotechar != '\0') {
    auto count = std::distance(it_begin, it_end) - 1;

    auto filtered_count = count;
    bool quotation = false;
    for (int i = 1; i < count; ++i) {
      if (h_data[h_row_offsets[i] - 1] == opts.quotechar) {
        quotation = !quotation;
        h_row_offsets[i] = static_cast<uint64_t>(-1);
        filtered_count--;
      } else if (quotation) {
        h_row_offsets[i] = static_cast<uint64_t>(-1);
        filtered_count--;
      }
    }
    if (filtered_count != count) {
      it_end = std::remove_if(it_begin, it_end, [](uint64_t pos) {
        return (pos == static_cast<uint64_t>(-1));
      });
    }
  }

  // Exclude the rows that are to be skipped from the start
  if (skip_rows != 0 && skip_rows < std::distance(it_begin, it_end)) {
    it_begin += skip_rows;
  }

  // Exclude the rows outside of requested range
  if (range_size != 0) {
    auto it = it_end - 1;
    while (it >= it_begin && *it > static_cast<uint64_t>(range_size)) {
      --it;
    }
    if ((it + 2) < it_end) {
      it_end = it + 2;
    }
  }

  // Exclude the rows without data
  if (opts.skipblanklines || opts.comment != '\0') {
    const auto newline = opts.skipblanklines ? opts.terminator : opts.comment;
    const auto comment = opts.comment != '\0' ? opts.comment : newline;
    const auto carriage =
        (opts.skipblanklines && opts.terminator == '\n') ? '\r' : comment;

    it_end = std::remove_if(it_begin, it_end, [=, &h_data](uint64_t pos) {
      return ((pos != h_size) &&
              (h_data[pos] == newline || h_data[pos] == comment ||
               h_data[pos] == carriage));
    });
  }

  // Exclude the rows before the header row (inclusive)
  if (std::distance(it_begin, it_end) > 1) {
    if (args_.header == -1) {
      header.assign(h_data + *(it_begin), h_data + *(it_begin + 1));
    } else {
      header.assign(h_data + *(it_begin + args_.header),
                    h_data + *(it_begin + args_.header + 1));
      it_begin += args_.header + 1;
    }
  }

  // Exclude the rows that exceed past the requested number
  if (num_rows >= 0 && num_rows < std::distance(it_begin, it_end)) {
    it_end = it_begin + num_rows + 1;
  }

  // Exclude the rows that are to be skipped from the end
  if (skip_end_rows != 0 && skip_end_rows < std::distance(it_begin, it_end)) {
    it_end -= skip_end_rows;
  }

  const uint64_t offset_start = *it_begin;
  const uint64_t offset_end = *(it_end - 1);

  // Copy out the row starts to use for row-column data parsing
  if (offset_start != offset_end) {
    if (offset_start != 0) {
      for (auto it = it_begin; it != it_end; ++it) {
        *it -= offset_start;
      }
    }
    CUDA_TRY(hipMemcpyAsync(row_offsets.data().get(), &(*it_begin),
                             std::distance(it_begin, it_end) * sizeof(uint64_t),
                             hipMemcpyHostToDevice));

    // Exclude the end-of-data row from number of rows with actual data
    num_records = std::distance(it_begin, it_end) - 1;
  }

  return std::make_pair(offset_start, offset_end);
}

std::pair<std::vector<gdf_dtype>, std::vector<gdf_dtype_extra_info>>
reader::Impl::gather_column_dtypes() {
  std::vector<gdf_dtype> dtypes;
  std::vector<gdf_dtype_extra_info> dtypes_extra_info;

  if (args_.dtype.empty()) {
    // If no input dtypes, default to info with TIME_UNIT_NONE
    dtypes_extra_info = vector<gdf_dtype_extra_info>(num_active_cols, gdf_dtype_extra_info{ TIME_UNIT_NONE });
    if (num_records == 0) {
      dtypes.resize(num_active_cols, GDF_STRING);
    } else {
      d_column_flags = h_column_flags;

      hostdevice_vector<column_parse::stats> column_stats(num_active_cols);
      CUDA_TRY(hipMemsetAsync(column_stats.device_ptr(), 0,
                               column_stats.memory_size()));
      CUDA_TRY(gpu::DetectCsvDataTypes(
          data.data(), row_offsets.data().get(), num_records, num_actual_cols,
          opts, d_column_flags.data().get(), column_stats.device_ptr()));
      CUDA_TRY(
          hipMemcpyAsync(column_stats.host_ptr(), column_stats.device_ptr(),
                          column_stats.memory_size(), hipMemcpyDeviceToHost));
      CUDA_TRY(hipStreamSynchronize(0));

      for (int col = 0; col < num_active_cols; col++) {
        unsigned long long countInt =
            column_stats[col].countInt8 + column_stats[col].countInt16 +
            column_stats[col].countInt32 + column_stats[col].countInt64;

        if (column_stats[col].countNULL == num_records) {
          // Entire column is NULL; allocate the smallest amount of memory
          dtypes.push_back(GDF_INT8);
        } else if (column_stats[col].countString > 0L) {
          dtypes.push_back(GDF_STRING);
        } else if (column_stats[col].countDateAndTime > 0L) {
          dtypes.push_back(GDF_DATE64);
        } else if (column_stats[col].countBool > 0L) {
          dtypes.push_back(GDF_BOOL8);
        } else if (column_stats[col].countFloat > 0L ||
                   (column_stats[col].countFloat == 0L && countInt > 0L &&
                    column_stats[col].countNULL > 0L)) {
          // The second condition has been added to conform to
          // PANDAS which states that a column of integers with
          // a single NULL record need to be treated as floats.
          dtypes.push_back(GDF_FLOAT64);
        } else {
          // All other integers are stored as 64-bit to conform to PANDAS
          dtypes.push_back(GDF_INT64);
        }
      }
    }
  } else {
    const bool is_dict = std::all_of(
        args_.dtype.begin(), args_.dtype.end(),
        [](const auto &s) { return s.find(':') != std::string::npos; });

    if (!is_dict) {
      if (args_.dtype.size() == 1) {
        // If it's a single dtype, assign that dtype to all active columns
        gdf_dtype dtype_;
        gdf_dtype_extra_info dtype_info_;
        column_parse::flags col_flags_;
        std::tie(dtype_, dtype_info_, col_flags_) = get_dtype_info(args_.dtype[0]);
        dtypes.resize(num_active_cols, dtype_);
        dtypes_extra_info.resize(num_active_cols, dtype_info_);
        for (int col = 0; col < num_actual_cols; col++) {
          h_column_flags[col] |= col_flags_;
        }
        CUDF_EXPECTS(dtypes.back() != GDF_invalid, "Unsupported data type");
      } else {
        // If it's a list, assign dtypes to active columns in the given order
        CUDF_EXPECTS(static_cast<int>(args_.dtype.size()) >= num_actual_cols,
                     "Must specify data types for all columns");

        auto dtype_ = std::back_inserter(dtypes);
        auto dtype_info_ = std::back_inserter(dtypes_extra_info);

        for (int col = 0; col < num_actual_cols; col++) {
          if (h_column_flags[col] & column_parse::enabled) {
            column_parse::flags col_flags_;
            std::tie(dtype_, dtype_info_, col_flags_) = get_dtype_info(args_.dtype[col]);
            h_column_flags[col] |= col_flags_;
            CUDF_EXPECTS(dtypes.back() != GDF_invalid, "Unsupported data type");
          }
        }
      }
    } else {
      // Translate vector of `name : dtype` strings to map
      // NOTE: Incoming pairs can be out-of-order from column names in dataset
      std::unordered_map<std::string, std::string> col_type_map;
      for (const auto &pair : args_.dtype) {
        const auto pos = pair.find_last_of(':');
        const auto name = pair.substr(0, pos);
        const auto dtype = pair.substr(pos + 1, pair.size());
        col_type_map[name] = dtype;
      }

      auto dtype_ = std::back_inserter(dtypes);
      auto dtype_info_ = std::back_inserter(dtypes_extra_info);

      for (int col = 0; col < num_actual_cols; col++) {
        if (h_column_flags[col] & column_parse::enabled) {
          CUDF_EXPECTS(col_type_map.find(col_names[col]) != col_type_map.end(),
                       "Must specify data types for all active columns");
          column_parse::flags col_flags_;
          std::tie(dtype_, dtype_info_, col_flags_) = get_dtype_info(col_type_map[col_names[col]]);
          h_column_flags[col] |= col_flags_;
          CUDF_EXPECTS(dtypes.back() != GDF_invalid, "Unsupported data type");
        }
      }
    }
  }

  return std::make_pair(dtypes, dtypes_extra_info);
}

void reader::Impl::decode_data(const std::vector<gdf_column_wrapper> &columns) {
  thrust::host_vector<gdf_dtype> h_dtypes(num_active_cols);
  thrust::host_vector<void*> h_data(num_active_cols);
  thrust::host_vector<gdf_valid_type*> h_valid(num_active_cols);

  for (int i = 0; i < num_active_cols; ++i) {
    h_dtypes[i] = columns[i]->dtype;
    h_data[i] = columns[i]->data;
    h_valid[i] = columns[i]->valid;
  }

  rmm::device_vector<gdf_dtype> d_dtypes = h_dtypes;
  rmm::device_vector<void*> d_data = h_data;
  rmm::device_vector<gdf_valid_type*> d_valid = h_valid;
  rmm::device_vector<gdf_size_type> d_valid_counts(num_active_cols, 0);
  d_column_flags = h_column_flags;

  CUDA_TRY(gpu::DecodeCsvColumnData(
      data.data(), row_offsets.data().get(), num_records, num_actual_cols, opts,
      d_column_flags.data().get(), d_dtypes.data().get(), d_data.data().get(),
      d_valid.data().get(), d_valid_counts.data().get()));
  CUDA_TRY(hipStreamSynchronize(0));

  thrust::host_vector<gdf_size_type> h_valid_counts = d_valid_counts;
  for (int i = 0; i < num_active_cols; ++i) {
    columns[i]->null_count = columns[i]->size - h_valid_counts[i];
  }
}

reader::Impl::Impl(std::unique_ptr<datasource> source,
                   std::string filepath, reader_options const &options)
    : source_(std::move(source)), filepath_(filepath), args_(options) {
  num_actual_cols = args_.names.size();
  num_active_cols = args_.names.size();

  if (args_.delim_whitespace) {
    opts.delimiter = ' ';
    opts.multi_delimiter = true;
  } else {
    opts.delimiter = args_.delimiter;
    opts.multi_delimiter = false;
  }
  opts.terminator = args_.lineterminator;
  if (args_.quotechar != '\0' && args_.quoting != QUOTE_NONE) {
    opts.quotechar = args_.quotechar;
    opts.keepquotes = false;
    opts.doublequote = args_.doublequote;
  } else {
    opts.quotechar = '\0';
    opts.keepquotes = true;
    opts.doublequote = false;
  }
  opts.skipblanklines = args_.skip_blank_lines;
  opts.comment = args_.comment;
  opts.dayfirst = args_.dayfirst;
  opts.decimal = args_.decimal;
  opts.thousands = args_.thousands;
  CUDF_EXPECTS(opts.decimal != opts.delimiter,
               "Decimal point cannot be the same as the delimiter");
  CUDF_EXPECTS(opts.thousands != opts.delimiter,
               "Thousands separator cannot be the same as the delimiter");

  compression_type_ = infer_compression_type(
      args_.compression, filepath,
      {{"gz", "gzip"}, {"zip", "zip"}, {"bz2", "bz2"}, {"xz", "xz"}});

  // Handle user-defined booleans, whereby field data are substituted with
  // true/false values; for numeric dtypes, they are mapped to 1/0 respectively
  std::vector<string> true_values{"True", "TRUE", "true"};
  true_values.insert(true_values.end(), args_.true_values.begin(),
                     args_.true_values.end());
  d_trueTrie = createSerializedTrie(true_values);
  opts.trueValuesTrie = d_trueTrie.data().get();

  std::vector<string> false_values{"False", "FALSE", "false"};
  false_values.insert(false_values.end(), args_.false_values.begin(),
                      args_.false_values.end());
  d_falseTrie = createSerializedTrie(false_values);
  opts.falseValuesTrie = d_falseTrie.data().get();

  // Handle user-defined NA values, whereby field data is treated as invalid
  if (args_.na_filter && (args_.keep_default_na || !args_.na_values.empty())) {
    std::vector<string> na_values{"#N/A",     "#N/A N/A", "#NA",  "-1.#IND",
                                  "-1.#QNAN", "-NaN",     "-nan", "1.#IND",
                                  "1.#QNAN",  "N/A",      "NA",   "NULL",
                                  "NaN",      "n/a",      "nan",  "null"};
    if (!args_.keep_default_na) {
      na_values.clear();
    }
    na_values.insert(na_values.end(), args_.na_values.begin(),
                     args_.na_values.end());
    d_naTrie = createSerializedTrie(na_values);
    opts.naValuesTrie = d_naTrie.data().get();
  }
}

reader::reader(std::string filepath, reader_options const &options)
    : impl_(std::make_unique<Impl>(nullptr, filepath, options)) {
  // Delay actual instantiation of data source until read to allow for
  // partial memory mapping of file using byte ranges
}

reader::reader(const char *buffer, size_t length, reader_options const &options)
    : impl_(std::make_unique<Impl>(datasource::create(buffer, length), "",
                                   options)) {}

reader::reader(std::shared_ptr<arrow::io::RandomAccessFile> file,
               reader_options const &options)
    : impl_(std::make_unique<Impl>(datasource::create(file), "", options)) {}

table reader::read() { return impl_->read(0, 0, 0, 0, -1); }

table reader::read_byte_range(size_t offset, size_t size) {
  return impl_->read(offset, size, 0, 0, -1);
}

table reader::read_rows(gdf_size_type num_skip_header,
                        gdf_size_type num_skip_footer, gdf_size_type num_rows) {
  CUDF_EXPECTS(num_rows == -1 || num_skip_footer == 0,
               "Cannot use both `num_rows` and `num_skip_footer`");

  return impl_->read(0, 0, num_skip_header, num_skip_footer, num_rows);
}

reader::~reader() = default;

}  // namespace csv
}  // namespace io
}  // namespace cudf
