#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

/**
 * @file reader_impl.cu
 * @brief cuDF-IO CSV reader class implementation
 **/

#include "reader_impl.hpp"

#include <algorithm>
#include <iostream>
#include <numeric>
#include <tuple>
#include <unordered_map>

#include <cudf/strings/replace.hpp>
#include <cudf/table/table.hpp>
#include <cudf/utilities/error.hpp>

#include <io/comp/io_uncomp.h>
#include <io/utilities/parsing_utils.cuh>
#include <io/utilities/type_conversion.cuh>

using std::string;
using std::vector;

namespace cudf {
namespace io {
namespace detail {
namespace csv {
using namespace cudf::io::csv;
using namespace cudf::io;

/**
 * @brief Estimates the maximum expected length or a row, based on the number
 * of columns
 *
 * If the number of columns is not available, it will return a value large
 * enough for most use cases
 *
 * @param[in] num_columns Number of columns in the CSV file (optional)
 *
 * @return Estimated maximum size of a row, in bytes
 **/
constexpr size_t calculateMaxRowSize(int num_columns = 0) noexcept
{
  constexpr size_t max_row_bytes = 16 * 1024;  // 16KB
  constexpr size_t column_bytes  = 64;
  constexpr size_t base_padding  = 1024;  // 1KB
  if (num_columns == 0) {
    // Use flat size if the number of columns is not known
    return max_row_bytes;
  } else {
    // Expand the size based on the number of columns, if available
    return base_padding + num_columns * column_bytes;
  }
}

/**
 * @brief Translates a dtype string and returns its dtype enumeration and any
 * extended dtype flags that are supported by cuIO. Often, this is a column
 * with the same underlying dtype the basic types, but with different parsing
 * interpretations.
 *
 * @param[in] dtype String containing the basic or extended dtype
 *
 * @return Tuple of data_type and flags
 */
std::tuple<data_type, column_parse::flags> get_dtype_info(const std::string &dtype)
{
  if (dtype == "hex" || dtype == "hex64") {
    return std::make_tuple(data_type{cudf::type_id::INT64}, column_parse::as_hexadecimal);
  }
  if (dtype == "hex32") {
    return std::make_tuple(data_type{cudf::type_id::INT32}, column_parse::as_hexadecimal);
  }

  return std::make_tuple(convert_string_to_dtype(dtype), column_parse::as_default);
}

/**
 * @brief Removes the first and Last quote in the string
 */
string removeQuotes(string str, char quotechar)
{
  // Exclude first and last quotation char
  const size_t first_quote = str.find(quotechar);
  if (first_quote != string::npos) { str.erase(first_quote, 1); }
  const size_t last_quote = str.rfind(quotechar);
  if (last_quote != string::npos) { str.erase(last_quote, 1); }

  return str;
}

/**
 * @brief Parse the first row to set the column names in the raw_csv parameter.
 * The first row can be either the header row, or the first data row
 */
std::vector<std::string> setColumnNames(std::vector<char> const &header,
                                        ParseOptions const &opts,
                                        int header_row,
                                        std::string prefix)
{
  std::vector<std::string> col_names;

  // If there is only a single character then it would be the terminator
  if (header.size() <= 1) { return col_names; }

  std::vector<char> first_row = header;
  int num_cols                = 0;

  bool quotation = false;
  for (size_t pos = 0, prev = 0; pos < first_row.size(); ++pos) {
    // Flip the quotation flag if current character is a quotechar
    if (first_row[pos] == opts.quotechar) {
      quotation = !quotation;
    }
    // Check if end of a column/row
    else if (pos == first_row.size() - 1 || (!quotation && first_row[pos] == opts.terminator) ||
             (!quotation && first_row[pos] == opts.delimiter)) {
      // This is the header, add the column name
      if (header_row >= 0) {
        // Include the current character, in case the line is not terminated
        int col_name_len = pos - prev + 1;
        // Exclude the delimiter/terminator is present
        if (first_row[pos] == opts.delimiter || first_row[pos] == opts.terminator) {
          --col_name_len;
        }
        // Also exclude '\r' character at the end of the column name if it's
        // part of the terminator
        if (col_name_len > 0 && opts.terminator == '\n' && first_row[pos] == '\n' &&
            first_row[pos - 1] == '\r') {
          --col_name_len;
        }

        const string new_col_name(first_row.data() + prev, col_name_len);
        col_names.push_back(removeQuotes(new_col_name, opts.quotechar));

        // Stop parsing when we hit the line terminator; relevant when there is
        // a blank line following the header. In this case, first_row includes
        // multiple line terminators at the end, as the new recStart belongs to
        // a line that comes after the blank line(s)
        if (!quotation && first_row[pos] == opts.terminator) { break; }
      } else {
        // This is the first data row, add the automatically generated name
        col_names.push_back(prefix + std::to_string(num_cols));
      }
      num_cols++;

      // Skip adjacent delimiters if delim_whitespace is set
      while (opts.multi_delimiter && pos < first_row.size() && first_row[pos] == opts.delimiter &&
             first_row[pos + 1] == opts.delimiter) {
        ++pos;
      }
      prev = pos + 1;
    }
  }

  return col_names;
}

table_with_metadata reader::impl::read(size_t range_offset,
                                       size_t range_size,
                                       int skip_rows,
                                       int skip_end_rows,
                                       int num_rows,
                                       hipStream_t stream)
{
  std::vector<std::unique_ptr<column>> out_columns;
  table_metadata metadata;

  if (range_offset > 0 || range_size > 0) {
    CUDF_EXPECTS(compression_type_ == "none",
                 "Reading compressed data using `byte range` is unsupported");
  }
  size_t map_range_size = 0;
  if (range_size != 0) {
    const auto num_columns = std::max(args_.names.size(), args_.dtype.size());
    map_range_size         = range_size + calculateMaxRowSize(num_columns);
  }

  // Support delayed opening of the file if using memory mapping datasource
  // This allows only mapping of a subset of the file if using byte range
  if (source_ == nullptr) {
    assert(!filepath_.empty());
    source_ = datasource::create(filepath_, range_offset, map_range_size);
  }

  // Return an empty dataframe if no data and no column metadata to process
  if (source_->is_empty() && (args_.names.empty() || args_.dtype.empty())) {
    return {std::make_unique<table>(std::move(out_columns)), std::move(metadata)};
  }

  // Transfer source data to GPU
  if (!source_->is_empty()) {
    const char *h_uncomp_data = nullptr;
    size_t h_uncomp_size      = 0;

    auto data_size = (map_range_size != 0) ? map_range_size : source_->size();
    auto buffer    = source_->host_read(range_offset, data_size);

    std::vector<char> h_uncomp_data_owner;
    if (compression_type_ == "none") {
      // Do not use the owner vector here to avoid extra copy
      h_uncomp_data = reinterpret_cast<const char *>(buffer->data());
      h_uncomp_size = buffer->size();
    } else {
      getUncompressedHostData(reinterpret_cast<const char *>(buffer->data()),
                              buffer->size(),
                              compression_type_,
                              h_uncomp_data_owner);
      h_uncomp_data = h_uncomp_data_owner.data();
      h_uncomp_size = h_uncomp_data_owner.size();
    }
    // None of the parameters for row selection is used, we are parsing the entire file
    const bool load_whole_file = range_offset == 0 && range_size == 0 && skip_rows <= 0 &&
                                 skip_end_rows <= 0 && num_rows == -1;

    // With byte range, find the start of the first data row
    size_t const data_start_offset =
      (range_offset != 0) ? find_first_row_start(h_uncomp_data, h_uncomp_size) : 0;

    // TODO: Allow parsing the header outside the mapped range
    CUDF_EXPECTS((range_offset == 0 || args_.header < 0),
                 "byte_range offset with header not supported");

    // Gather row offsets
    gather_row_offsets(h_uncomp_data,
                       h_uncomp_size,
                       data_start_offset,
                       (range_size) ? range_size : h_uncomp_size,
                       (skip_rows > 0) ? skip_rows : 0,
                       num_rows,
                       load_whole_file,
                       stream);

    // Exclude the rows that are to be skipped from the end
    if (skip_end_rows > 0 && static_cast<size_t>(skip_end_rows) < row_offsets.size()) {
      row_offsets.resize(row_offsets.size() - skip_end_rows);
    }

    // Exclude the end-of-data row from number of rows with actual data
    num_records = row_offsets.size();
    num_records -= (num_records > 0);
  } else {
    num_records = 0;
  }

  // Check if the user gave us a list of column names
  if (not args_.names.empty()) {
    h_column_flags.resize(args_.names.size(), column_parse::enabled);
    col_names = args_.names;
  } else {
    col_names = setColumnNames(header, opts, args_.header, args_.prefix);

    num_actual_cols = num_active_cols = col_names.size();

    h_column_flags.resize(num_actual_cols, column_parse::enabled);

    // Rename empty column names to "Unnamed: col_index"
    for (size_t col_idx = 0; col_idx < col_names.size(); ++col_idx) {
      if (col_names[col_idx].empty()) {
        col_names[col_idx] = string("Unnamed: ") + std::to_string(col_idx);
      }
    }

    // Looking for duplicates
    std::unordered_map<string, int> col_names_histogram;
    for (auto &col_name : col_names) {
      // Operator [] inserts a default-initialized value if the given key is not
      // present
      if (++col_names_histogram[col_name] > 1) {
        if (args_.mangle_dupe_cols) {
          // Rename duplicates of column X as X.1, X.2, ...; First appearance
          // stays as X
          col_name += "." + std::to_string(col_names_histogram[col_name] - 1);
        } else {
          // All duplicate columns will be ignored; First appearance is parsed
          const auto idx      = &col_name - col_names.data();
          h_column_flags[idx] = column_parse::disabled;
        }
      }
    }

    // Update the number of columns to be processed, if some might have been
    // removed
    if (!args_.mangle_dupe_cols) { num_active_cols = col_names_histogram.size(); }
  }

  // User can specify which columns should be parsed
  if (!args_.use_cols_indexes.empty() || !args_.use_cols_names.empty()) {
    std::fill(h_column_flags.begin(), h_column_flags.end(), column_parse::disabled);

    for (const auto index : args_.use_cols_indexes) {
      h_column_flags[index] = column_parse::enabled;
    }
    num_active_cols = args_.use_cols_indexes.size();

    for (const auto name : args_.use_cols_names) {
      const auto it = std::find(col_names.begin(), col_names.end(), name);
      if (it != col_names.end()) {
        h_column_flags[it - col_names.begin()] = column_parse::enabled;
        num_active_cols++;
      }
    }
  }

  // User can specify which columns should be inferred as datetime
  if (!args_.infer_date_indexes.empty() || !args_.infer_date_names.empty()) {
    for (const auto index : args_.infer_date_indexes) {
      h_column_flags[index] |= column_parse::as_datetime;
    }

    for (const auto name : args_.infer_date_names) {
      auto it = std::find(col_names.begin(), col_names.end(), name);
      if (it != col_names.end()) {
        h_column_flags[it - col_names.begin()] |= column_parse::as_datetime;
      }
    }
  }

  // Return empty table rather than exception if nothing to load
  if (num_active_cols == 0) {
    return {std::make_unique<table>(std::move(out_columns)), std::move(metadata)};
  }

  std::vector<data_type> column_types = gather_column_types(stream);

  // Alloc output; columns' data memory is still expected for empty dataframe
  std::vector<column_buffer> out_buffers;
  out_buffers.reserve(column_types.size());
  for (int col = 0, active_col = 0; col < num_actual_cols; ++col) {
    if (h_column_flags[col] & column_parse::enabled) {
      // Replace EMPTY dtype with STRING
      if (column_types[active_col].id() == type_id::EMPTY) {
        column_types[active_col] = data_type{type_id::STRING};
      }
      const bool is_final_allocation = column_types[active_col].id() != type_id::STRING;
      out_buffers.emplace_back(column_types[active_col],
                               num_records,
                               true,
                               stream,
                               is_final_allocation ? mr_ : rmm::mr::get_default_resource());
      metadata.column_names.emplace_back(col_names[col]);
      active_col++;
    }
  }

  out_columns.reserve(column_types.size());
  if (num_records != 0) {
    decode_data(column_types, out_buffers, stream);

    for (size_t i = 0; i < column_types.size(); ++i) {
      if (column_types[i].id() == type_id::STRING && opts.quotechar != '\0' &&
          opts.doublequote == true) {
        // PANDAS' default behavior of enabling doublequote for two consecutive
        // quotechars in quoted fields results in reduction to a single quotechar
        // TODO: Would be much more efficient to perform this operation in-place
        // during the conversion stage
        const std::string quotechar(1, opts.quotechar);
        const std::string dblquotechar(2, opts.quotechar);
        std::unique_ptr<column> col = make_strings_column(out_buffers[i]._strings, stream);
        out_columns.emplace_back(
          cudf::strings::replace(col->view(), dblquotechar, quotechar, -1, mr_));
      } else {
        out_columns.emplace_back(
          make_column(column_types[i], num_records, out_buffers[i], stream, mr_));
      }
    }
  } else {
    // Create empty columns
    for (size_t i = 0; i < column_types.size(); ++i) {
      out_columns.emplace_back(make_empty_column(column_types[i]));
    }
  }
  return {std::make_unique<table>(std::move(out_columns)), std::move(metadata)};
}

size_t reader::impl::find_first_row_start(const char *h_data, size_t h_size)
{
  // For now, look for the first terminator (assume the first terminator isn't within a quote)
  // TODO: Attempt to infer this from the data
  size_t pos = 0;
  while (pos < h_size && h_data[pos] != opts.terminator) { ++pos; }
  return std::min(pos + 1, h_size);
}

void reader::impl::gather_row_offsets(const char *h_data,
                                      size_t h_size,
                                      size_t range_begin,
                                      size_t range_end,
                                      size_t skip_rows,
                                      int64_t num_rows,
                                      bool load_whole_file,
                                      hipStream_t stream)
{
  constexpr size_t max_chunk_bytes = 64 * 1024 * 1024;  // 64MB
  size_t buffer_size               = std::min(max_chunk_bytes, h_size);
  size_t max_blocks =
    std::max<size_t>((buffer_size / cudf::io::csv::gpu::rowofs_block_bytes) + 1, 2);
  hostdevice_vector<uint64_t> row_ctx(max_blocks);
  size_t buffer_pos  = std::min(range_begin - std::min(range_begin, sizeof(char)), h_size);
  size_t pos         = std::min(range_begin, h_size);
  size_t header_rows = (args_.header >= 0) ? args_.header + 1 : 0;
  uint64_t ctx       = 0;

  // For compatibility with the previous parser, a row is considered in-range if the
  // previous row terminator is within the given range
  range_end += (range_end < h_size);
  data_.resize(0);
  row_offsets.resize(0);
  data_.reserve((load_whole_file) ? h_size : std::min(buffer_size * 2, h_size));
  do {
    size_t target_pos = std::min(pos + max_chunk_bytes, h_size);
    size_t chunk_size = target_pos - pos;

    data_.insert(data_.end(), h_data + buffer_pos + data_.size(), h_data + target_pos);

    // Pass 1: Count the potential number of rows in each character block for each
    // possible parser state at the beginning of the block.
    uint32_t num_blocks = cudf::io::csv::gpu::gather_row_offsets(row_ctx.device_ptr(),
                                                                 nullptr,
                                                                 data_.data().get(),
                                                                 chunk_size,
                                                                 pos,
                                                                 buffer_pos,
                                                                 h_size,
                                                                 range_begin,
                                                                 range_end,
                                                                 skip_rows,
                                                                 0,
                                                                 opts,
                                                                 stream);
    CUDA_TRY(hipMemcpyAsync(row_ctx.host_ptr(),
                             row_ctx.device_ptr(),
                             num_blocks * sizeof(uint64_t),
                             hipMemcpyDeviceToHost,
                             stream));
    CUDA_TRY(hipStreamSynchronize(stream));
    // Sum up the rows in each character block, selecting the row count that
    // corresponds to the current input context. Also stores the now known input
    // context per character block that will be needed by the second pass.
    for (uint32_t i = 0; i < num_blocks; i++) {
      uint64_t ctx_next = cudf::io::csv::gpu::select_row_context(ctx, row_ctx[i]);
      row_ctx[i]        = ctx;
      ctx               = ctx_next;
    }
    size_t total_rows = ctx >> 2;
    if (total_rows > skip_rows) {
      // At least one row in range in this batch
      size_t num_row_offsets = total_rows - skip_rows;
      row_offsets.resize(num_row_offsets);
      CUDA_TRY(hipMemcpyAsync(row_ctx.device_ptr(),
                               row_ctx.host_ptr(),
                               num_blocks * sizeof(uint64_t),
                               hipMemcpyHostToDevice,
                               stream));
      // Pass 2: Output row offsets
      cudf::io::csv::gpu::gather_row_offsets(row_ctx.device_ptr(),
                                             row_offsets.data().get(),
                                             data_.data().get(),
                                             chunk_size,
                                             pos,
                                             buffer_pos,
                                             h_size,
                                             range_begin,
                                             range_end,
                                             skip_rows,
                                             num_row_offsets,
                                             opts,
                                             stream);
      // With byte range, we want to keep only one row out of the specified range
      if (range_end < h_size) {
        CUDA_TRY(hipMemcpyAsync(row_ctx.host_ptr(),
                                 row_ctx.device_ptr(),
                                 num_blocks * sizeof(uint64_t),
                                 hipMemcpyDeviceToHost,
                                 stream));
        CUDA_TRY(hipStreamSynchronize(stream));
        size_t rows_out_of_range = 0;
        for (uint32_t i = 0; i < num_blocks; i++) { rows_out_of_range += row_ctx[i]; }
        if (rows_out_of_range != 0) {
          // Keep one row out of range (used to infer length of previous row)
          num_row_offsets -= std::min(rows_out_of_range - 1, num_row_offsets);
          row_offsets.resize(num_row_offsets);
          // Implies we reached the end of the range
          break;
        }
      }
      // num_rows does not include blank rows
      if (num_rows >= 0) {
        if (num_row_offsets > header_rows + static_cast<size_t>(num_rows)) {
          size_t num_blanks =
            cudf::io::csv::gpu::count_blank_rows(row_offsets, data_, opts, stream);
          if (num_row_offsets - num_blanks > header_rows + static_cast<size_t>(num_rows)) {
            // Got the desired number of rows
            break;
          }
        }
      }
    } else {
      // Discard data (all rows below skip_rows), keeping one character for history
      size_t discard_bytes = std::max(data_.size(), sizeof(char)) - sizeof(char);
      if (discard_bytes != 0) {
        data_.erase(data_.begin(), data_.begin() + discard_bytes);
        buffer_pos += discard_bytes;
      }
    }
    pos = target_pos;
  } while (pos < h_size);

  // Eliminate blank rows
  if (row_offsets.size() != 0) {
    cudf::io::csv::gpu::remove_blank_rows(row_offsets, data_, opts, stream);
  }
  // Remove header rows and extract header
  const size_t header_row_index = std::max<size_t>(header_rows, 1) - 1;
  if (header_row_index + 1 < row_offsets.size()) {
    CUDA_TRY(hipMemcpyAsync(row_ctx.host_ptr(),
                             row_offsets.data().get() + header_row_index,
                             2 * sizeof(uint64_t),
                             hipMemcpyDeviceToHost,
                             stream));
    CUDA_TRY(hipStreamSynchronize(stream));
    const auto header_start = buffer_pos + row_ctx[0];
    const auto header_end   = buffer_pos + row_ctx[1];
    CUDF_EXPECTS(header_start <= header_end && header_end <= h_size, "Invalid csv header location");
    header.assign(h_data + header_start, h_data + header_end);
    if (header_rows > 0) {
      row_offsets.erase(row_offsets.begin(), row_offsets.begin() + header_rows);
    }
  }
  // Apply num_rows limit
  if (num_rows >= 0) { row_offsets.resize(std::min<size_t>(row_offsets.size(), num_rows + 1)); }
}

std::vector<data_type> reader::impl::gather_column_types(hipStream_t stream)
{
  std::vector<data_type> dtypes;

  if (args_.dtype.empty()) {
    if (num_records == 0) {
      dtypes.resize(num_active_cols, data_type{type_id::EMPTY});
    } else {
      d_column_flags = h_column_flags;

      hostdevice_vector<column_parse::stats> column_stats(num_active_cols);
      CUDA_TRY(hipMemsetAsync(column_stats.device_ptr(), 0, column_stats.memory_size(), stream));
      CUDA_TRY(cudf::io::csv::gpu::DetectColumnTypes(data_.data().get(),
                                                     row_offsets.data().get(),
                                                     num_records,
                                                     num_actual_cols,
                                                     opts,
                                                     d_column_flags.data().get(),
                                                     column_stats.device_ptr(),
                                                     stream));
      CUDA_TRY(hipMemcpyAsync(column_stats.host_ptr(),
                               column_stats.device_ptr(),
                               column_stats.memory_size(),
                               hipMemcpyDeviceToHost,
                               stream));
      CUDA_TRY(hipStreamSynchronize(stream));

      for (int col = 0; col < num_active_cols; col++) {
        unsigned long long countInt = column_stats[col].countInt8 + column_stats[col].countInt16 +
                                      column_stats[col].countInt32 + column_stats[col].countInt64;

        if (column_stats[col].countNULL == num_records) {
          // Entire column is NULL; allocate the smallest amount of memory
          dtypes.emplace_back(cudf::type_id::INT8);
        } else if (column_stats[col].countString > 0L) {
          dtypes.emplace_back(cudf::type_id::STRING);
        } else if (column_stats[col].countDateAndTime > 0L) {
          dtypes.emplace_back(cudf::type_id::TIMESTAMP_NANOSECONDS);
        } else if (column_stats[col].countBool > 0L) {
          dtypes.emplace_back(cudf::type_id::BOOL8);
        } else if (column_stats[col].countFloat > 0L ||
                   (column_stats[col].countFloat == 0L && countInt > 0L &&
                    column_stats[col].countNULL > 0L)) {
          // The second condition has been added to conform to
          // PANDAS which states that a column of integers with
          // a single NULL record need to be treated as floats.
          dtypes.emplace_back(cudf::type_id::FLOAT64);
        } else {
          // All other integers are stored as 64-bit to conform to PANDAS
          dtypes.emplace_back(cudf::type_id::INT64);
        }
      }
    }
  } else {
    const bool is_dict = std::all_of(args_.dtype.begin(), args_.dtype.end(), [](const auto &s) {
      return s.find(':') != std::string::npos;
    });

    if (!is_dict) {
      if (args_.dtype.size() == 1) {
        // If it's a single dtype, assign that dtype to all active columns
        data_type dtype_;
        column_parse::flags col_flags_;
        std::tie(dtype_, col_flags_) = get_dtype_info(args_.dtype[0]);
        dtypes.resize(num_active_cols, dtype_);
        for (int col = 0; col < num_actual_cols; col++) { h_column_flags[col] |= col_flags_; }
        CUDF_EXPECTS(dtypes.back().id() != cudf::type_id::EMPTY, "Unsupported data type");
      } else {
        // If it's a list, assign dtypes to active columns in the given order
        CUDF_EXPECTS(static_cast<int>(args_.dtype.size()) >= num_actual_cols,
                     "Must specify data types for all columns");

        auto dtype_ = std::back_inserter(dtypes);

        for (int col = 0; col < num_actual_cols; col++) {
          if (h_column_flags[col] & column_parse::enabled) {
            column_parse::flags col_flags_;
            std::tie(dtype_, col_flags_) = get_dtype_info(args_.dtype[col]);
            h_column_flags[col] |= col_flags_;
            CUDF_EXPECTS(dtypes.back().id() != cudf::type_id::EMPTY, "Unsupported data type");
          }
        }
      }
    } else {
      // Translate vector of `name : dtype` strings to map
      // NOTE: Incoming pairs can be out-of-order from column names in dataset
      std::unordered_map<std::string, std::string> col_type_map;
      for (const auto &pair : args_.dtype) {
        const auto pos     = pair.find_last_of(':');
        const auto name    = pair.substr(0, pos);
        const auto dtype   = pair.substr(pos + 1, pair.size());
        col_type_map[name] = dtype;
      }

      auto dtype_ = std::back_inserter(dtypes);

      for (int col = 0; col < num_actual_cols; col++) {
        if (h_column_flags[col] & column_parse::enabled) {
          CUDF_EXPECTS(col_type_map.find(col_names[col]) != col_type_map.end(),
                       "Must specify data types for all active columns");
          column_parse::flags col_flags_;
          std::tie(dtype_, col_flags_) = get_dtype_info(col_type_map[col_names[col]]);
          h_column_flags[col] |= col_flags_;
          CUDF_EXPECTS(dtypes.back().id() != cudf::type_id::EMPTY, "Unsupported data type");
        }
      }
    }
  }

  if (args_.timestamp_type.id() != cudf::type_id::EMPTY) {
    for (auto &type : dtypes) {
      if (cudf::is_timestamp(type)) { type = args_.timestamp_type; }
    }
  }

  return dtypes;
}

void reader::impl::decode_data(const std::vector<data_type> &column_types,
                               std::vector<column_buffer> &out_buffers,
                               hipStream_t stream)
{
  thrust::host_vector<void *> h_data(num_active_cols);
  thrust::host_vector<bitmask_type *> h_valid(num_active_cols);

  for (int i = 0; i < num_active_cols; ++i) {
    h_data[i]  = out_buffers[i].data();
    h_valid[i] = out_buffers[i].null_mask();
  }

  rmm::device_vector<data_type> d_dtypes(column_types);
  rmm::device_vector<void *> d_data          = h_data;
  rmm::device_vector<bitmask_type *> d_valid = h_valid;
  d_column_flags                             = h_column_flags;

  CUDA_TRY(cudf::io::csv::gpu::DecodeRowColumnData(data_.data().get(),
                                                   row_offsets.data().get(),
                                                   num_records,
                                                   num_actual_cols,
                                                   opts,
                                                   d_column_flags.data().get(),
                                                   d_dtypes.data().get(),
                                                   d_data.data().get(),
                                                   d_valid.data().get(),
                                                   stream));
  CUDA_TRY(hipStreamSynchronize(stream));

  for (int i = 0; i < num_active_cols; ++i) { out_buffers[i].null_count() = UNKNOWN_NULL_COUNT; }
}

reader::impl::impl(std::unique_ptr<datasource> source,
                   std::string filepath,
                   reader_options const &options,
                   rmm::mr::device_memory_resource *mr)
  : source_(std::move(source)), mr_(mr), filepath_(filepath), args_(options)
{
  num_actual_cols = args_.names.size();
  num_active_cols = args_.names.size();

  if (args_.delim_whitespace) {
    opts.delimiter       = ' ';
    opts.multi_delimiter = true;
  } else {
    opts.delimiter       = args_.delimiter;
    opts.multi_delimiter = false;
  }
  opts.terminator = args_.lineterminator;
  if (args_.quotechar != '\0' && args_.quoting != quote_style::NONE) {
    opts.quotechar   = args_.quotechar;
    opts.keepquotes  = false;
    opts.doublequote = args_.doublequote;
  } else {
    opts.quotechar   = '\0';
    opts.keepquotes  = true;
    opts.doublequote = false;
  }
  opts.skipblanklines = args_.skip_blank_lines;
  opts.comment        = args_.comment;
  opts.dayfirst       = args_.dayfirst;
  opts.decimal        = args_.decimal;
  opts.thousands      = args_.thousands;
  CUDF_EXPECTS(opts.decimal != opts.delimiter, "Decimal point cannot be the same as the delimiter");
  CUDF_EXPECTS(opts.thousands != opts.delimiter,
               "Thousands separator cannot be the same as the delimiter");

  compression_type_ = infer_compression_type(
    args_.compression, filepath, {{"gz", "gzip"}, {"zip", "zip"}, {"bz2", "bz2"}, {"xz", "xz"}});

  // Handle user-defined false values, whereby field data is substituted with a
  // boolean true or numeric `1` value
  if (args_.true_values.size() != 0) {
    d_trueTrie          = createSerializedTrie(args_.true_values);
    opts.trueValuesTrie = d_trueTrie.data().get();
  }

  // Handle user-defined false values, whereby field data is substituted with a
  // boolean false or numeric `0` value
  if (args_.false_values.size() != 0) {
    d_falseTrie          = createSerializedTrie(args_.false_values);
    opts.falseValuesTrie = d_falseTrie.data().get();
  }

  // Handle user-defined N/A values, whereby field data is treated as null
  if (args_.na_values.size() != 0) {
    d_naTrie          = createSerializedTrie(args_.na_values);
    opts.naValuesTrie = d_naTrie.data().get();
  }
}

// Forward to implementation
reader::reader(std::string filepath,
               reader_options const &options,
               rmm::mr::device_memory_resource *mr)
  : _impl(std::make_unique<impl>(nullptr, filepath, options, mr))
{
  // Delay actual instantiation of data source until read to allow for
  // partial memory mapping of file using byte ranges
}

// Forward to implementation
reader::reader(std::unique_ptr<cudf::io::datasource> source,
               reader_options const &options,
               rmm::mr::device_memory_resource *mr)
  : _impl(std::make_unique<impl>(std::move(source), "", options, mr))
{
}

// Destructor within this translation unit
reader::~reader() = default;

// Forward to implementation
table_with_metadata reader::read_all(hipStream_t stream)
{
  return _impl->read(0, 0, 0, 0, -1, stream);
}

// Forward to implementation
table_with_metadata reader::read_byte_range(size_t offset, size_t size, hipStream_t stream)
{
  return _impl->read(offset, size, 0, 0, -1, stream);
}

// Forward to implementation
table_with_metadata reader::read_rows(size_type num_skip_header,
                                      size_type num_skip_footer,
                                      size_type num_rows,
                                      hipStream_t stream)
{
  CUDF_EXPECTS(num_rows == -1 || num_skip_footer == 0,
               "Cannot use both `num_rows` and `num_skip_footer`");

  return _impl->read(0, 0, num_skip_header, num_skip_footer, num_rows, stream);
}

}  // namespace csv
}  // namespace detail
}  // namespace io
}  // namespace cudf
