#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019-2021, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

/**
 * @file reader_impl.cu
 * @brief cuDF-IO CSV reader class implementation
 */

#include "reader_impl.hpp"

#include <io/comp/io_uncomp.h>
#include <io/utilities/parsing_utils.cuh>
#include <io/utilities/type_conversion.cuh>

#include <cudf/detail/utilities/cuda.cuh>
#include <cudf/detail/utilities/vector_factories.hpp>
#include <cudf/io/types.hpp>
#include <cudf/strings/replace.hpp>
#include <cudf/table/table.hpp>
#include <cudf/utilities/error.hpp>
#include <cudf/utilities/span.hpp>

#include <rmm/cuda_stream_view.hpp>

#include <algorithm>
#include <iostream>
#include <numeric>
#include <tuple>
#include <unordered_map>

using std::string;
using std::vector;

using cudf::device_span;
using cudf::host_span;
using cudf::detail::make_device_uvector_async;

namespace {
/**
 * @brief Helper class to support inline-overloading for all of a variant's alternative types
 */
template <class... Ts>
struct VisitorOverload : Ts... {
  using Ts::operator()...;
};
template <class... Ts>
VisitorOverload(Ts...)->VisitorOverload<Ts...>;
}  // namespace

namespace cudf {
namespace io {
namespace detail {
namespace csv {
using namespace cudf::io::csv;
using namespace cudf::io;

/**
 * @brief Estimates the maximum expected length or a row, based on the number
 * of columns
 *
 * If the number of columns is not available, it will return a value large
 * enough for most use cases
 *
 * @param[in] num_columns Number of columns in the CSV file (optional)
 *
 * @return Estimated maximum size of a row, in bytes
 */
constexpr size_t calculateMaxRowSize(int num_columns = 0) noexcept
{
  constexpr size_t max_row_bytes = 16 * 1024;  // 16KB
  constexpr size_t column_bytes  = 64;
  constexpr size_t base_padding  = 1024;  // 1KB
  if (num_columns == 0) {
    // Use flat size if the number of columns is not known
    return max_row_bytes;
  } else {
    // Expand the size based on the number of columns, if available
    return base_padding + num_columns * column_bytes;
  }
}

/**
 * @brief Translates a dtype string and returns its dtype enumeration and any
 * extended dtype flags that are supported by cuIO. Often, this is a column
 * with the same underlying dtype the basic types, but with different parsing
 * interpretations.
 *
 * @param[in] dtype String containing the basic or extended dtype
 *
 * @return Tuple of data_type and flags
 */
std::tuple<data_type, column_parse::flags> get_dtype_info(const std::string &dtype)
{
  if (dtype == "hex" || dtype == "hex64") {
    return std::make_tuple(data_type{cudf::type_id::INT64}, column_parse::as_hexadecimal);
  }
  if (dtype == "hex32") {
    return std::make_tuple(data_type{cudf::type_id::INT32}, column_parse::as_hexadecimal);
  }

  return std::make_tuple(convert_string_to_dtype(dtype), column_parse::as_default);
}

/**
 * @brief Removes the first and Last quote in the string
 */
string removeQuotes(string str, char quotechar)
{
  // Exclude first and last quotation char
  const size_t first_quote = str.find(quotechar);
  if (first_quote != string::npos) { str.erase(first_quote, 1); }
  const size_t last_quote = str.rfind(quotechar);
  if (last_quote != string::npos) { str.erase(last_quote, 1); }

  return str;
}

/**
 * @brief Parse the first row to set the column names in the raw_csv parameter.
 * The first row can be either the header row, or the first data row
 */
std::vector<std::string> setColumnNames(std::vector<char> const &header,
                                        parse_options_view const &opts,
                                        int header_row,
                                        std::string prefix)
{
  std::vector<std::string> col_names;

  // If there is only a single character then it would be the terminator
  if (header.size() <= 1) { return col_names; }

  std::vector<char> first_row = header;
  int num_cols                = 0;

  bool quotation = false;
  for (size_t pos = 0, prev = 0; pos < first_row.size(); ++pos) {
    // Flip the quotation flag if current character is a quotechar
    if (first_row[pos] == opts.quotechar) {
      quotation = !quotation;
    }
    // Check if end of a column/row
    else if (pos == first_row.size() - 1 || (!quotation && first_row[pos] == opts.terminator) ||
             (!quotation && first_row[pos] == opts.delimiter)) {
      // This is the header, add the column name
      if (header_row >= 0) {
        // Include the current character, in case the line is not terminated
        int col_name_len = pos - prev + 1;
        // Exclude the delimiter/terminator is present
        if (first_row[pos] == opts.delimiter || first_row[pos] == opts.terminator) {
          --col_name_len;
        }
        // Also exclude '\r' character at the end of the column name if it's
        // part of the terminator
        if (col_name_len > 0 && opts.terminator == '\n' && first_row[pos] == '\n' &&
            first_row[pos - 1] == '\r') {
          --col_name_len;
        }

        const string new_col_name(first_row.data() + prev, col_name_len);
        col_names.push_back(removeQuotes(new_col_name, opts.quotechar));

        // Stop parsing when we hit the line terminator; relevant when there is
        // a blank line following the header. In this case, first_row includes
        // multiple line terminators at the end, as the new recStart belongs to
        // a line that comes after the blank line(s)
        if (!quotation && first_row[pos] == opts.terminator) { break; }
      } else {
        // This is the first data row, add the automatically generated name
        col_names.push_back(prefix + std::to_string(num_cols));
      }
      num_cols++;

      // Skip adjacent delimiters if delim_whitespace is set
      while (opts.multi_delimiter && pos < first_row.size() && first_row[pos] == opts.delimiter &&
             first_row[pos + 1] == opts.delimiter) {
        ++pos;
      }
      prev = pos + 1;
    }
  }

  return col_names;
}

template <typename C>
void erase_except_last(C &container, rmm::cuda_stream_view stream)
{
  cudf::detail::device_single_thread(
    [span = device_span<typename C::value_type>{container}] __device__() mutable {
      span.front() = span.back();
    },
    stream);
  container.resize(1, stream);
}

std::pair<rmm::device_uvector<char>, reader::impl::selected_rows_offsets>
reader::impl::select_data_and_row_offsets(rmm::cuda_stream_view stream)
{
  auto range_offset  = opts_.get_byte_range_offset();
  auto range_size    = opts_.get_byte_range_size();
  auto skip_rows     = opts_.get_skiprows();
  auto skip_end_rows = opts_.get_skipfooter();
  auto num_rows      = opts_.get_nrows();

  if (range_offset > 0 || range_size > 0) {
    CUDF_EXPECTS(compression_type_ == "none",
                 "Reading compressed data using `byte range` is unsupported");
  }
  size_t map_range_size = 0;
  if (range_size != 0) {
    auto num_given_dtypes =
      std::visit([](const auto &dtypes) { return dtypes.size(); }, opts_.get_dtypes());
    const auto num_columns = std::max(opts_.get_names().size(), num_given_dtypes);
    map_range_size         = range_size + calculateMaxRowSize(num_columns);
  }

  // Support delayed opening of the file if using memory mapping datasource
  // This allows only mapping of a subset of the file if using byte range
  if (source_ == nullptr) {
    assert(!filepath_.empty());
    source_ = datasource::create(filepath_, range_offset, map_range_size);
  }

  // Transfer source data to GPU
  if (!source_->is_empty()) {
    auto data_size = (map_range_size != 0) ? map_range_size : source_->size();
    auto buffer    = source_->host_read(range_offset, data_size);

    auto h_data = host_span<char const>(  //
      reinterpret_cast<const char *>(buffer->data()),
      buffer->size());

    std::vector<char> h_uncomp_data_owner;

    if (compression_type_ != "none") {
      h_uncomp_data_owner = get_uncompressed_data(h_data, compression_type_);
      h_data              = h_uncomp_data_owner;
    }
    // None of the parameters for row selection is used, we are parsing the entire file
    const bool load_whole_file = range_offset == 0 && range_size == 0 && skip_rows <= 0 &&
                                 skip_end_rows <= 0 && num_rows == -1;

    // With byte range, find the start of the first data row
    size_t const data_start_offset = (range_offset != 0) ? find_first_row_start(h_data) : 0;

    // TODO: Allow parsing the header outside the mapped range
    CUDF_EXPECTS((range_offset == 0 || opts_.get_header() < 0),
                 "byte_range offset with header not supported");

    // Gather row offsets
    auto data_row_offsets =
      load_data_and_gather_row_offsets(h_data,
                                       data_start_offset,
                                       (range_size) ? range_size : h_data.size(),
                                       (skip_rows > 0) ? skip_rows : 0,
                                       num_rows,
                                       load_whole_file,
                                       stream);
    auto &row_offsets = data_row_offsets.second;
    // Exclude the rows that are to be skipped from the end
    if (skip_end_rows > 0 && static_cast<size_t>(skip_end_rows) < row_offsets.size()) {
      row_offsets.shrink(row_offsets.size() - skip_end_rows);
    }
    return data_row_offsets;
  }
  return {rmm::device_uvector<char>{0, stream}, selected_rows_offsets{stream}};
}

table_with_metadata reader::impl::read(rmm::cuda_stream_view stream)
{
  auto const data_row_offsets = select_data_and_row_offsets(stream);
  auto const &data            = data_row_offsets.first;
  auto const &row_offsets     = data_row_offsets.second;

  // Exclude the end-of-data row from number of rows with actual data
  num_records_ = std::max(row_offsets.size(), 1ul) - 1;

  // Check if the user gave us a list of column names
  if (not opts_.get_names().empty()) {
    column_flags_.resize(opts_.get_names().size(), column_parse::enabled);
    col_names_ = opts_.get_names();
  } else {
    col_names_ = setColumnNames(header_, opts.view(), opts_.get_header(), opts_.get_prefix());

    num_actual_cols_ = num_active_cols_ = col_names_.size();

    column_flags_.resize(num_actual_cols_, column_parse::enabled);

    // Rename empty column names to "Unnamed: col_index"
    for (size_t col_idx = 0; col_idx < col_names_.size(); ++col_idx) {
      if (col_names_[col_idx].empty()) {
        col_names_[col_idx] = string("Unnamed: ") + std::to_string(col_idx);
      }
    }

    // Looking for duplicates
    std::unordered_map<string, int> col_names_histogram;
    for (auto &col_name : col_names_) {
      // Operator [] inserts a default-initialized value if the given key is not
      // present
      if (++col_names_histogram[col_name] > 1) {
        if (opts_.is_enabled_mangle_dupe_cols()) {
          // Rename duplicates of column X as X.1, X.2, ...; First appearance
          // stays as X
          col_name += "." + std::to_string(col_names_histogram[col_name] - 1);
        } else {
          // All duplicate columns will be ignored; First appearance is parsed
          const auto idx     = &col_name - col_names_.data();
          column_flags_[idx] = column_parse::disabled;
        }
      }
    }

    // Update the number of columns to be processed, if some might have been
    // removed
    if (!opts_.is_enabled_mangle_dupe_cols()) { num_active_cols_ = col_names_histogram.size(); }
  }

  // User can specify which columns should be parsed
  if (!opts_.get_use_cols_indexes().empty() || !opts_.get_use_cols_names().empty()) {
    std::fill(column_flags_.begin(), column_flags_.end(), column_parse::disabled);

    for (const auto index : opts_.get_use_cols_indexes()) {
      column_flags_[index] = column_parse::enabled;
    }
    num_active_cols_ = opts_.get_use_cols_indexes().size();

    for (const auto &name : opts_.get_use_cols_names()) {
      const auto it = std::find(col_names_.begin(), col_names_.end(), name);
      if (it != col_names_.end()) {
        column_flags_[it - col_names_.begin()] = column_parse::enabled;
        num_active_cols_++;
      }
    }
  }

  // User can specify which columns should be inferred as datetime
  if (!opts_.get_infer_date_indexes().empty() || !opts_.get_infer_date_names().empty()) {
    for (const auto index : opts_.get_infer_date_indexes()) {
      column_flags_[index] |= column_parse::as_datetime;
    }

    for (const auto &name : opts_.get_infer_date_names()) {
      auto it = std::find(col_names_.begin(), col_names_.end(), name);
      if (it != col_names_.end()) {
        column_flags_[it - col_names_.begin()] |= column_parse::as_datetime;
      }
    }
  }

  // Return empty table rather than exception if nothing to load
  if (num_active_cols_ == 0) { return {std::make_unique<table>(), {}}; }

  auto metadata    = table_metadata{};
  auto out_columns = std::vector<std::unique_ptr<cudf::column>>();

  bool has_to_infer_column_types =
    std::visit([](const auto &dtypes) { return dtypes.empty(); }, opts_.get_dtypes());

  std::vector<data_type> column_types;
  if (has_to_infer_column_types) {
    column_types = infer_column_types(data, row_offsets, stream);
  } else {
    column_types =
      std::visit(VisitorOverload{
                   [&](const std::vector<data_type> &data_types) { return data_types; },
                   [&](const std::vector<string> &dtypes) { return parse_column_types(dtypes); }},
                 opts_.get_dtypes());
  }

  out_columns.reserve(column_types.size());

  if (num_records_ != 0) {
    auto out_buffers = decode_data(data, row_offsets, column_types, stream);
    for (size_t i = 0; i < column_types.size(); ++i) {
      metadata.column_names.emplace_back(out_buffers[i].name);
      if (column_types[i].id() == type_id::STRING && opts.quotechar != '\0' &&
          opts.doublequote == true) {
        // PANDAS' default behavior of enabling doublequote for two consecutive
        // quotechars in quoted fields results in reduction to a single quotechar
        // TODO: Would be much more efficient to perform this operation in-place
        // during the conversion stage
        const std::string quotechar(1, opts.quotechar);
        const std::string dblquotechar(2, opts.quotechar);
        std::unique_ptr<column> col = cudf::make_strings_column(*out_buffers[i]._strings, stream);
        out_columns.emplace_back(
          cudf::strings::replace(col->view(), dblquotechar, quotechar, -1, mr_));
      } else {
        out_columns.emplace_back(make_column(out_buffers[i], nullptr, stream, mr_));
      }
    }
  } else {
    // Create empty columns
    for (size_t i = 0; i < column_types.size(); ++i) {
      out_columns.emplace_back(make_empty_column(column_types[i]));
    }
    // Handle empty metadata
    for (int col = 0; col < num_actual_cols_; ++col) {
      if (column_flags_[col] & column_parse::enabled) {
        metadata.column_names.emplace_back(col_names_[col]);
      }
    }
  }
  return {std::make_unique<table>(std::move(out_columns)), std::move(metadata)};
}

size_t reader::impl::find_first_row_start(host_span<char const> data)
{
  // For now, look for the first terminator (assume the first terminator isn't within a quote)
  // TODO: Attempt to infer this from the data
  size_t pos = 0;
  while (pos < data.size() && data[pos] != opts.terminator) { ++pos; }
  return std::min(pos + 1, data.size());
}

std::pair<rmm::device_uvector<char>, reader::impl::selected_rows_offsets>
reader::impl::load_data_and_gather_row_offsets(host_span<char const> data,
                                               size_t range_begin,
                                               size_t range_end,
                                               size_t skip_rows,
                                               int64_t num_rows,
                                               bool load_whole_file,
                                               rmm::cuda_stream_view stream)
{
  constexpr size_t max_chunk_bytes = 64 * 1024 * 1024;  // 64MB
  size_t buffer_size               = std::min(max_chunk_bytes, data.size());
  size_t max_blocks =
    std::max<size_t>((buffer_size / cudf::io::csv::gpu::rowofs_block_bytes) + 1, 2);
  hostdevice_vector<uint64_t> row_ctx(max_blocks);
  size_t buffer_pos  = std::min(range_begin - std::min(range_begin, sizeof(char)), data.size());
  size_t pos         = std::min(range_begin, data.size());
  size_t header_rows = (opts_.get_header() >= 0) ? opts_.get_header() + 1 : 0;
  uint64_t ctx       = 0;

  // For compatibility with the previous parser, a row is considered in-range if the
  // previous row terminator is within the given range
  range_end += (range_end < data.size());

  // Reserve memory by allocating and then resetting the size
  rmm::device_uvector<char> d_data{
    (load_whole_file) ? data.size() : std::min(buffer_size * 2, data.size()), stream};
  d_data.resize(0, stream);
  rmm::device_uvector<uint64_t> all_row_offsets{0, stream};
  do {
    size_t target_pos = std::min(pos + max_chunk_bytes, data.size());
    size_t chunk_size = target_pos - pos;

    auto const previous_data_size = d_data.size();
    d_data.resize(target_pos - buffer_pos, stream);
    CUDA_TRY(hipMemcpyAsync(d_data.begin() + previous_data_size,
                             data.begin() + buffer_pos + previous_data_size,
                             target_pos - buffer_pos - previous_data_size,
                             hipMemcpyDefault,
                             stream.value()));

    // Pass 1: Count the potential number of rows in each character block for each
    // possible parser state at the beginning of the block.
    uint32_t num_blocks = cudf::io::csv::gpu::gather_row_offsets(opts.view(),
                                                                 row_ctx.device_ptr(),
                                                                 device_span<uint64_t>(),
                                                                 d_data,
                                                                 chunk_size,
                                                                 pos,
                                                                 buffer_pos,
                                                                 data.size(),
                                                                 range_begin,
                                                                 range_end,
                                                                 skip_rows,
                                                                 stream);
    CUDA_TRY(hipMemcpyAsync(row_ctx.host_ptr(),
                             row_ctx.device_ptr(),
                             num_blocks * sizeof(uint64_t),
                             hipMemcpyDeviceToHost,
                             stream.value()));
    stream.synchronize();

    // Sum up the rows in each character block, selecting the row count that
    // corresponds to the current input context. Also stores the now known input
    // context per character block that will be needed by the second pass.
    for (uint32_t i = 0; i < num_blocks; i++) {
      uint64_t ctx_next = cudf::io::csv::gpu::select_row_context(ctx, row_ctx[i]);
      row_ctx[i]        = ctx;
      ctx               = ctx_next;
    }
    size_t total_rows = ctx >> 2;
    if (total_rows > skip_rows) {
      // At least one row in range in this batch
      all_row_offsets.resize(total_rows - skip_rows, stream);

      CUDA_TRY(hipMemcpyAsync(row_ctx.device_ptr(),
                               row_ctx.host_ptr(),
                               num_blocks * sizeof(uint64_t),
                               hipMemcpyHostToDevice,
                               stream.value()));

      // Pass 2: Output row offsets
      cudf::io::csv::gpu::gather_row_offsets(opts.view(),
                                             row_ctx.device_ptr(),
                                             all_row_offsets,
                                             d_data,
                                             chunk_size,
                                             pos,
                                             buffer_pos,
                                             data.size(),
                                             range_begin,
                                             range_end,
                                             skip_rows,
                                             stream);
      // With byte range, we want to keep only one row out of the specified range
      if (range_end < data.size()) {
        CUDA_TRY(hipMemcpyAsync(row_ctx.host_ptr(),
                                 row_ctx.device_ptr(),
                                 num_blocks * sizeof(uint64_t),
                                 hipMemcpyDeviceToHost,
                                 stream.value()));
        stream.synchronize();

        size_t rows_out_of_range = 0;
        for (uint32_t i = 0; i < num_blocks; i++) { rows_out_of_range += row_ctx[i]; }
        if (rows_out_of_range != 0) {
          // Keep one row out of range (used to infer length of previous row)
          auto new_row_offsets_size =
            all_row_offsets.size() - std::min(rows_out_of_range - 1, all_row_offsets.size());
          all_row_offsets.resize(new_row_offsets_size, stream);
          // Implies we reached the end of the range
          break;
        }
      }
      // num_rows does not include blank rows
      if (num_rows >= 0) {
        if (all_row_offsets.size() > header_rows + static_cast<size_t>(num_rows)) {
          size_t num_blanks =
            cudf::io::csv::gpu::count_blank_rows(opts.view(), d_data, all_row_offsets, stream);
          if (all_row_offsets.size() - num_blanks > header_rows + static_cast<size_t>(num_rows)) {
            // Got the desired number of rows
            break;
          }
        }
      }
    } else {
      // Discard data (all rows below skip_rows), keeping one character for history
      size_t discard_bytes = std::max(d_data.size(), sizeof(char)) - sizeof(char);
      if (discard_bytes != 0) {
        erase_except_last(d_data, stream);
        buffer_pos += discard_bytes;
      }
    }
    pos = target_pos;
  } while (pos < data.size());

  auto const non_blank_row_offsets =
    io::csv::gpu::remove_blank_rows(opts.view(), d_data, all_row_offsets, stream);
  auto row_offsets = selected_rows_offsets{std::move(all_row_offsets), non_blank_row_offsets};

  // Remove header rows and extract header
  const size_t header_row_index = std::max<size_t>(header_rows, 1) - 1;
  if (header_row_index + 1 < row_offsets.size()) {
    CUDA_TRY(hipMemcpyAsync(row_ctx.host_ptr(),
                             row_offsets.data() + header_row_index,
                             2 * sizeof(uint64_t),
                             hipMemcpyDeviceToHost,
                             stream.value()));
    stream.synchronize();

    const auto header_start = buffer_pos + row_ctx[0];
    const auto header_end   = buffer_pos + row_ctx[1];
    CUDF_EXPECTS(header_start <= header_end && header_end <= data.size(),
                 "Invalid csv header location");
    header_.assign(data.begin() + header_start, data.begin() + header_end);
    if (header_rows > 0) { row_offsets.erase_first_n(header_rows); }
  }
  // Apply num_rows limit
  if (num_rows >= 0 && static_cast<size_t>(num_rows) < row_offsets.size() - 1) {
    row_offsets.shrink(num_rows + 1);
  }
  return {std::move(d_data), std::move(row_offsets)};
}

std::vector<data_type> reader::impl::infer_column_types(device_span<char const> data,
                                                        device_span<uint64_t const> row_offsets,
                                                        rmm::cuda_stream_view stream)
{
  std::vector<data_type> dtypes;
  if (num_records_ == 0) {
    dtypes.resize(num_active_cols_, data_type{type_id::EMPTY});
  } else {
    auto column_stats =
      cudf::io::csv::gpu::detect_column_types(opts.view(),
                                              data,
                                              make_device_uvector_async(column_flags_, stream),
                                              row_offsets,
                                              num_active_cols_,
                                              stream);

    stream.synchronize();

    for (int col = 0; col < num_active_cols_; col++) {
      unsigned long long int_count_total = column_stats[col].big_int_count +
                                           column_stats[col].negative_small_int_count +
                                           column_stats[col].positive_small_int_count;

      if (column_stats[col].null_count == num_records_) {
        // Entire column is NULL; allocate the smallest amount of memory
        dtypes.emplace_back(cudf::type_id::INT8);
      } else if (column_stats[col].string_count > 0L) {
        dtypes.emplace_back(cudf::type_id::STRING);
      } else if (column_stats[col].datetime_count > 0L) {
        dtypes.emplace_back(cudf::type_id::TIMESTAMP_NANOSECONDS);
      } else if (column_stats[col].bool_count > 0L) {
        dtypes.emplace_back(cudf::type_id::BOOL8);
      } else if (column_stats[col].float_count > 0L ||
                 (column_stats[col].float_count == 0L && int_count_total > 0L &&
                  column_stats[col].null_count > 0L)) {
        // The second condition has been added to conform to
        // PANDAS which states that a column of integers with
        // a single NULL record need to be treated as floats.
        dtypes.emplace_back(cudf::type_id::FLOAT64);
      } else if (column_stats[col].big_int_count == 0) {
        dtypes.emplace_back(cudf::type_id::INT64);
      } else if (column_stats[col].big_int_count != 0 &&
                 column_stats[col].negative_small_int_count != 0) {
        dtypes.emplace_back(cudf::type_id::STRING);
      } else {
        // Integers are stored as 64-bit to conform to PANDAS
        dtypes.emplace_back(cudf::type_id::UINT64);
      }
    }
  }

  if (opts_.get_timestamp_type().id() != cudf::type_id::EMPTY) {
    for (auto &type : dtypes) {
      if (cudf::is_timestamp(type)) { type = opts_.get_timestamp_type(); }
    }
  }

  for (size_t i = 0; i < dtypes.size(); i++) {
    // Replace EMPTY dtype with STRING
    if (dtypes[i].id() == type_id::EMPTY) { dtypes[i] = data_type{type_id::STRING}; }
  }

  return dtypes;
}

std::vector<data_type> reader::impl::parse_column_types(
  const std::vector<std::string> &types_as_strings)
{
  std::vector<data_type> dtypes;

  const bool is_dict = std::all_of(types_as_strings.begin(),
                                   types_as_strings.end(),
                                   [](const auto &s) { return s.find(':') != std::string::npos; });

  if (!is_dict) {
    if (types_as_strings.size() == 1) {
      // If it's a single dtype, assign that dtype to all active columns
      data_type dtype_;
      column_parse::flags col_flags_;
      std::tie(dtype_, col_flags_) = get_dtype_info(types_as_strings[0]);
      dtypes.resize(num_active_cols_, dtype_);
      for (int col = 0; col < num_actual_cols_; col++) { column_flags_[col] |= col_flags_; }
      CUDF_EXPECTS(dtypes.back().id() != cudf::type_id::EMPTY, "Unsupported data type");
    } else {
      // If it's a list, assign dtypes to active columns in the given order
      CUDF_EXPECTS(static_cast<int>(types_as_strings.size()) >= num_actual_cols_,
                   "Must specify data types for all columns");

      auto dtype_ = std::back_inserter(dtypes);

      for (int col = 0; col < num_actual_cols_; col++) {
        if (column_flags_[col] & column_parse::enabled) {
          column_parse::flags col_flags_;
          std::tie(dtype_, col_flags_) = get_dtype_info(types_as_strings[col]);
          column_flags_[col] |= col_flags_;
          CUDF_EXPECTS(dtypes.back().id() != cudf::type_id::EMPTY, "Unsupported data type");
        }
      }
    }
  } else {
    // Translate vector of `name : dtype` strings to map
    // NOTE: Incoming pairs can be out-of-order from column names in dataset
    std::unordered_map<std::string, std::string> col_type_map;
    for (const auto &pair : types_as_strings) {
      const auto pos     = pair.find_last_of(':');
      const auto name    = pair.substr(0, pos);
      const auto dtype   = pair.substr(pos + 1, pair.size());
      col_type_map[name] = dtype;
    }

    auto dtype_ = std::back_inserter(dtypes);

    for (int col = 0; col < num_actual_cols_; col++) {
      if (column_flags_[col] & column_parse::enabled) {
        CUDF_EXPECTS(col_type_map.find(col_names_[col]) != col_type_map.end(),
                     "Must specify data types for all active columns");
        column_parse::flags col_flags_;
        std::tie(dtype_, col_flags_) = get_dtype_info(col_type_map[col_names_[col]]);
        column_flags_[col] |= col_flags_;
        CUDF_EXPECTS(dtypes.back().id() != cudf::type_id::EMPTY, "Unsupported data type");
      }
    }
  }

  if (opts_.get_timestamp_type().id() != cudf::type_id::EMPTY) {
    for (auto &type : dtypes) {
      if (cudf::is_timestamp(type)) { type = opts_.get_timestamp_type(); }
    }
  }

  for (size_t i = 0; i < dtypes.size(); i++) {
    // Replace EMPTY dtype with STRING
    if (dtypes[i].id() == type_id::EMPTY) { dtypes[i] = data_type{type_id::STRING}; }
  }

  return dtypes;
}

std::vector<column_buffer> reader::impl::decode_data(device_span<char const> data,
                                                     device_span<uint64_t const> row_offsets,
                                                     host_span<data_type const> column_types,
                                                     rmm::cuda_stream_view stream)
{
  // Alloc output; columns' data memory is still expected for empty dataframe
  std::vector<column_buffer> out_buffers;
  out_buffers.reserve(column_types.size());

  for (int col = 0, active_col = 0; col < num_actual_cols_; ++col) {
    if (column_flags_[col] & column_parse::enabled) {
      const bool is_final_allocation = column_types[active_col].id() != type_id::STRING;
      auto out_buffer =
        column_buffer(column_types[active_col],
                      num_records_,
                      true,
                      stream,
                      is_final_allocation ? mr_ : rmm::mr::get_current_device_resource());

      out_buffer.name         = col_names_[col];
      out_buffer.null_count() = UNKNOWN_NULL_COUNT;
      out_buffers.emplace_back(std::move(out_buffer));
      active_col++;
    }
  }

  thrust::host_vector<void *> h_data(num_active_cols_);
  thrust::host_vector<bitmask_type *> h_valid(num_active_cols_);

  for (int i = 0; i < num_active_cols_; ++i) {
    h_data[i]  = out_buffers[i].data();
    h_valid[i] = out_buffers[i].null_mask();
  }

  cudf::io::csv::gpu::decode_row_column_data(opts.view(),
                                             data,
                                             make_device_uvector_async(column_flags_, stream),
                                             row_offsets,
                                             make_device_uvector_async(column_types, stream),
                                             make_device_uvector_async(h_data, stream),
                                             make_device_uvector_async(h_valid, stream),
                                             stream);

  return out_buffers;
}

/**
 * @brief Create a serialized trie for N/A value matching, based on the options.
 */
cudf::detail::trie create_na_trie(char quotechar,
                                  csv_reader_options const &reader_opts,
                                  rmm::cuda_stream_view stream)
{
  // Default values to recognize as null values
  static std::vector<std::string> const default_na_values{"",
                                                          "#N/A",
                                                          "#N/A N/A",
                                                          "#NA",
                                                          "-1.#IND",
                                                          "-1.#QNAN",
                                                          "-NaN",
                                                          "-nan",
                                                          "1.#IND",
                                                          "1.#QNAN",
                                                          "<NA>",
                                                          "N/A",
                                                          "NA",
                                                          "NULL",
                                                          "NaN",
                                                          "n/a",
                                                          "nan",
                                                          "null"};

  if (!reader_opts.is_enabled_na_filter()) { return cudf::detail::trie(0, stream); }

  std::vector<std::string> na_values = reader_opts.get_na_values();
  if (reader_opts.is_enabled_keep_default_na()) {
    na_values.insert(na_values.end(), default_na_values.begin(), default_na_values.end());
  }

  // Pandas treats empty strings as N/A if empty fields are treated as N/A
  if (std::find(na_values.begin(), na_values.end(), "") != na_values.end()) {
    na_values.push_back(std::string(2, quotechar));
  }

  return cudf::detail::create_serialized_trie(na_values, stream);
}

parse_options make_parse_options(csv_reader_options const &reader_opts,
                                 rmm::cuda_stream_view stream)
{
  auto parse_opts = parse_options{};

  if (reader_opts.is_enabled_delim_whitespace()) {
    parse_opts.delimiter       = ' ';
    parse_opts.multi_delimiter = true;
  } else {
    parse_opts.delimiter       = reader_opts.get_delimiter();
    parse_opts.multi_delimiter = false;
  }

  parse_opts.terminator = reader_opts.get_lineterminator();

  if (reader_opts.get_quotechar() != '\0' && reader_opts.get_quoting() != quote_style::NONE) {
    parse_opts.quotechar   = reader_opts.get_quotechar();
    parse_opts.keepquotes  = false;
    parse_opts.doublequote = reader_opts.is_enabled_doublequote();
  } else {
    parse_opts.quotechar   = '\0';
    parse_opts.keepquotes  = true;
    parse_opts.doublequote = false;
  }

  parse_opts.skipblanklines = reader_opts.is_enabled_skip_blank_lines();
  parse_opts.comment        = reader_opts.get_comment();
  parse_opts.dayfirst       = reader_opts.is_enabled_dayfirst();
  parse_opts.decimal        = reader_opts.get_decimal();
  parse_opts.thousands      = reader_opts.get_thousands();

  CUDF_EXPECTS(parse_opts.decimal != parse_opts.delimiter,
               "Decimal point cannot be the same as the delimiter");
  CUDF_EXPECTS(parse_opts.thousands != parse_opts.delimiter,
               "Thousands separator cannot be the same as the delimiter");

  // Handle user-defined true values, whereby field data is substituted with a
  // boolean true or numeric `1` value
  if (reader_opts.get_true_values().size() != 0) {
    parse_opts.trie_true =
      cudf::detail::create_serialized_trie(reader_opts.get_true_values(), stream);
  }

  // Handle user-defined false values, whereby field data is substituted with a
  // boolean false or numeric `0` value
  if (reader_opts.get_false_values().size() != 0) {
    parse_opts.trie_false =
      cudf::detail::create_serialized_trie(reader_opts.get_false_values(), stream);
  }

  // Handle user-defined N/A values, whereby field data is treated as null
  parse_opts.trie_na = create_na_trie(parse_opts.quotechar, reader_opts, stream);

  return parse_opts;
}

reader::impl::impl(std::unique_ptr<datasource> source,
                   std::string filepath,
                   csv_reader_options const &options,
                   rmm::cuda_stream_view stream,
                   rmm::mr::device_memory_resource *mr)
  : mr_(mr), source_(std::move(source)), filepath_(filepath), opts_(options)
{
  num_actual_cols_ = opts_.get_names().size();
  num_active_cols_ = num_actual_cols_;

  compression_type_ =
    infer_compression_type(opts_.get_compression(),
                           filepath,
                           {{"gz", "gzip"}, {"zip", "zip"}, {"bz2", "bz2"}, {"xz", "xz"}});

  opts = make_parse_options(options, stream);
}

// Forward to implementation
reader::reader(std::vector<std::string> const &filepaths,
               csv_reader_options const &options,
               rmm::cuda_stream_view stream,
               rmm::mr::device_memory_resource *mr)
{
  CUDF_EXPECTS(filepaths.size() == 1, "Only a single source is currently supported.");
  // Delay actual instantiation of data source until read to allow for
  // partial memory mapping of file using byte ranges
  _impl = std::make_unique<impl>(nullptr, filepaths[0], options, stream, mr);
}

// Forward to implementation
reader::reader(std::vector<std::unique_ptr<cudf::io::datasource>> &&sources,
               csv_reader_options const &options,
               rmm::cuda_stream_view stream,
               rmm::mr::device_memory_resource *mr)
{
  CUDF_EXPECTS(sources.size() == 1, "Only a single source is currently supported.");
  _impl = std::make_unique<impl>(std::move(sources[0]), "", options, stream, mr);
}

// Destructor within this translation unit
reader::~reader() = default;

// Forward to implementation
table_with_metadata reader::read(rmm::cuda_stream_view stream) { return _impl->read(stream); }

}  // namespace csv
}  // namespace detail
}  // namespace io
}  // namespace cudf
