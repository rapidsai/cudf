#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

/**
 * @file reader_impl.cu
 * @brief cuDF-IO CSV reader class implementation
 **/

#include "reader_impl.hpp"

#include <algorithm>
#include <iostream>
#include <numeric>
#include <tuple>
#include <unordered_map>

#include "legacy/datetime_parser.cuh"
#include "legacy/type_conversion.cuh"

#include <utilities/legacy/cudf_utils.h>
#include <cudf/legacy/unary.hpp>
#include <cudf/table/table.hpp>
#include <cudf/utilities/error.hpp>

#include <nvstrings/NVStrings.h>

#include <io/comp/io_uncomp.h>
#include <io/utilities/parsing_utils.cuh>
#include <io/utilities/type_conversion.cuh>

using std::string;
using std::vector;

namespace cudf {
namespace experimental {
namespace io {
namespace detail {
namespace csv {

using namespace cudf::io::csv;
using namespace cudf::io;

/**---------------------------------------------------------------------------*
 * @brief Estimates the maximum expected length or a row, based on the number
 * of columns
 *
 * If the number of columns is not available, it will return a value large
 * enough for most use cases
 *
 * @param[in] num_columns Number of columns in the CSV file (optional)
 *
 * @return Estimated maximum size of a row, in bytes
 *---------------------------------------------------------------------------**/
constexpr size_t calculateMaxRowSize(int num_columns = 0) noexcept {
  constexpr size_t max_row_bytes = 16 * 1024;  // 16KB
  constexpr size_t column_bytes = 64;
  constexpr size_t base_padding = 1024;  // 1KB
  if (num_columns == 0) {
    // Use flat size if the number of columns is not known
    return max_row_bytes;
  } else {
    // Expand the size based on the number of columns, if available
    return base_padding + num_columns * column_bytes;
  }
}

/**
 * @brief Translates a dtype string and returns its dtype enumeration and any
 * extended dtype flags that are supported by cuIO. Often, this is a column
 * with the same underlying dtype the basic types, but with different parsing
 * interpretations.
 *
 * @param[in] dtype String containing the basic or extended dtype
 *
 * @return std::pair<gdf_dtype, column_parse::flags> Tuple of dtype and flags
 */
std::tuple<data_type, column_parse::flags> get_dtype_info(
    const std::string &dtype) {
  if (dtype == "hex" || dtype == "hex64") {
    return std::make_tuple(data_type{cudf::type_id::INT64},
                           column_parse::as_hexadecimal);
  }
  if (dtype == "hex32") {
    return std::make_tuple(data_type{cudf::type_id::INT32},
                           column_parse::as_hexadecimal);
  }

  return std::make_tuple(convert_string_to_dtype(dtype), column_parse::as_default);
}

/**
 * @brief Removes the first and Last quote in the string
 */
string removeQuotes(string str, char quotechar) {
  // Exclude first and last quotation char
  const size_t first_quote = str.find(quotechar);
  if (first_quote != string::npos) {
    str.erase(first_quote, 1);
  }
  const size_t last_quote = str.rfind(quotechar);
  if (last_quote != string::npos) {
    str.erase(last_quote, 1);
  }

  return str;
}

/**
 * @brief Parse the first row to set the column names in the raw_csv parameter.
 * The first row can be either the header row, or the first data row
 */
std::vector<std::string> setColumnNames(std::vector<char> const &header,
                                        ParseOptions const &opts,
                                        int header_row, std::string prefix) {
  std::vector<std::string> col_names;

  // If there is only a single character then it would be the terminator
  if (header.size() <= 1) {
    return col_names;
  }

  std::vector<char> first_row = header;
  int num_cols = 0;

  bool quotation = false;
  for (size_t pos = 0, prev = 0; pos < first_row.size(); ++pos) {
    // Flip the quotation flag if current character is a quotechar
    if (first_row[pos] == opts.quotechar) {
      quotation = !quotation;
    }
    // Check if end of a column/row
    else if (pos == first_row.size() - 1 ||
             (!quotation && first_row[pos] == opts.terminator) ||
             (!quotation && first_row[pos] == opts.delimiter)) {
      // This is the header, add the column name
      if (header_row >= 0) {
        // Include the current character, in case the line is not terminated
        int col_name_len = pos - prev + 1;
        // Exclude the delimiter/terminator is present
        if (first_row[pos] == opts.delimiter ||
            first_row[pos] == opts.terminator) {
          --col_name_len;
        }
        // Also exclude '\r' character at the end of the column name if it's
        // part of the terminator
        if (col_name_len > 0 && opts.terminator == '\n' &&
            first_row[pos] == '\n' && first_row[pos - 1] == '\r') {
          --col_name_len;
        }

        const string new_col_name(first_row.data() + prev, col_name_len);
        col_names.push_back(removeQuotes(new_col_name, opts.quotechar));

        // Stop parsing when we hit the line terminator; relevant when there is
        // a blank line following the header. In this case, first_row includes
        // multiple line terminators at the end, as the new recStart belongs to
        // a line that comes after the blank line(s)
        if (!quotation && first_row[pos] == opts.terminator) {
          break;
        }
      } else {
        // This is the first data row, add the automatically generated name
        col_names.push_back(prefix + std::to_string(num_cols));
      }
      num_cols++;

      // Skip adjacent delimiters if delim_whitespace is set
      while (opts.multi_delimiter && pos < first_row.size() &&
             first_row[pos] == opts.delimiter &&
             first_row[pos + 1] == opts.delimiter) {
        ++pos;
      }
      prev = pos + 1;
    }
  }

  return col_names;
}

table_with_metadata reader::impl::read(size_t range_offset,
                                       size_t range_size, int skip_rows,
                                       int skip_end_rows, int num_rows,
                                       hipStream_t stream) {
  std::vector<std::unique_ptr<column>> out_columns;
  table_metadata metadata;

  if (range_offset > 0 || range_size > 0) {
    CUDF_EXPECTS(compression_type_ == "none",
                 "Reading compressed data using `byte range` is unsupported");
  }
  size_t map_range_size = 0;
  if (range_size != 0) {
    const auto num_columns = std::max(args_.names.size(), args_.dtype.size());
    map_range_size = range_size + calculateMaxRowSize(num_columns);
  }

  // Support delayed opening of the file if using memory mapping datasource
  // This allows only mapping of a subset of the file if using byte range
  if (source_ == nullptr) {
    assert(!filepath_.empty());
    source_ = datasource::create(filepath_, range_offset, map_range_size);
  }

  // Return an empty dataframe if no data and no column metadata to process
  if (source_->empty() && (args_.names.empty() || args_.dtype.empty())) {
    return { std::make_unique<table>(std::move(out_columns)), std::move(metadata) };
  }

  // Transfer source data to GPU
  if (!source_->empty()) {
    const char *h_uncomp_data = nullptr;
    size_t h_uncomp_size = 0;

    auto data_size = (map_range_size != 0) ? map_range_size : source_->size();
    auto buffer = source_->get_buffer(range_offset, data_size);

    std::vector<char> h_uncomp_data_owner;
    if (compression_type_ == "none") {
      // Do not use the owner vector here to avoid extra copy
      h_uncomp_data = reinterpret_cast<const char *>(buffer->data());
      h_uncomp_size = buffer->size();
    } else {
      CUDF_EXPECTS(
          getUncompressedHostData(
              reinterpret_cast<const char *>(buffer->data()), buffer->size(),
              compression_type_, h_uncomp_data_owner) == GDF_SUCCESS,
          "Cannot decompress data");
      h_uncomp_data = h_uncomp_data_owner.data();
      h_uncomp_size = h_uncomp_data_owner.size();
    }

    gather_row_offsets(h_uncomp_data, h_uncomp_size, range_offset, stream);
    auto row_range = select_rows(h_uncomp_data, h_uncomp_size, range_size,
                                 skip_rows, skip_end_rows, num_rows, stream);

    data_size = row_range.second - row_range.first;
    CUDF_EXPECTS(data_size <= h_uncomp_size, "Row range exceeds data size");

    num_bits = (data_size + 63) / 64;
    data_ = rmm::device_buffer(h_uncomp_data + row_range.first, data_size);
  }

  // Check if the user gave us a list of column names
  if (not args_.names.empty()) {
    h_column_flags.resize(args_.names.size(), column_parse::enabled);
    col_names = args_.names;
  } else {
    col_names = setColumnNames(header, opts, args_.header, args_.prefix);

    num_actual_cols = num_active_cols = col_names.size();

    h_column_flags.resize(num_actual_cols, column_parse::enabled);

    // Rename empty column names to "Unnamed: col_index"
    for (size_t col_idx = 0; col_idx < col_names.size(); ++col_idx) {
      if (col_names[col_idx].empty()) {
        col_names[col_idx] = string("Unnamed: ") + std::to_string(col_idx);
      }
    }

    // Looking for duplicates
    std::unordered_map<string, int> col_names_histogram;
    for (auto &col_name : col_names) {
      // Operator [] inserts a default-initialized value if the given key is not
      // present
      if (++col_names_histogram[col_name] > 1) {
        if (args_.mangle_dupe_cols) {
          // Rename duplicates of column X as X.1, X.2, ...; First appearance
          // stays as X
          col_name += "." + std::to_string(col_names_histogram[col_name] - 1);
        } else {
          // All duplicate columns will be ignored; First appearance is parsed
          const auto idx = &col_name - col_names.data();
          h_column_flags[idx] = column_parse::disabled;
        }
      }
    }

    // Update the number of columns to be processed, if some might have been
    // removed
    if (!args_.mangle_dupe_cols) {
      num_active_cols = col_names_histogram.size();
    }
  }

  // User can specify which columns should be parsed
  if (!args_.use_cols_indexes.empty() || !args_.use_cols_names.empty()) {
    std::fill(h_column_flags.begin(), h_column_flags.end(),
              column_parse::disabled);

    for (const auto index : args_.use_cols_indexes) {
      h_column_flags[index] = column_parse::enabled;
    }
    num_active_cols = args_.use_cols_indexes.size();

    for (const auto name : args_.use_cols_names) {
      const auto it = std::find(col_names.begin(), col_names.end(), name);
      if (it != col_names.end()) {
        h_column_flags[it - col_names.begin()] = column_parse::enabled;
        num_active_cols++;
      }
    }
  }

  // User can specify which columns should be inferred as datetime
  if (!args_.infer_date_indexes.empty() || !args_.infer_date_names.empty()) {
    for (const auto index : args_.infer_date_indexes) {
      h_column_flags[index] |= column_parse::as_datetime;
    }

    for (const auto name : args_.infer_date_names) {
      auto it = std::find(col_names.begin(), col_names.end(), name);
      if (it != col_names.end()) {
        h_column_flags[it - col_names.begin()] |= column_parse::as_datetime;
      }
    }
  }

  // Return empty table rather than exception if nothing to load
  if (num_active_cols == 0) {
    return { std::make_unique<table>(std::move(out_columns)), std::move(metadata) };
  }

  std::vector<data_type> column_types = gather_column_types(stream);

  // Alloc output; columns' data memory is still expected for empty dataframe
  std::vector<column_buffer> out_buffers;
  for (int col = 0, active_col = 0; col < num_actual_cols; ++col) {
    if (h_column_flags[col] & column_parse::enabled) {
      out_buffers.emplace_back(column_types[active_col], num_records, true,
                               stream, mr_);
      metadata.column_names.emplace_back(col_names[col]);
      active_col++;
    }
  }

  if (num_records != 0) {
    decode_data(column_types, out_buffers, stream);
  }

  for (size_t i = 0; i < column_types.size(); ++i) {
    out_columns.emplace_back(
        make_column(column_types[i], num_records, out_buffers[i]));
  }

  // TODO: String columns need to be reworked to actually copy characters in
  // kernel to allow skipping quotation characters
  /*for (auto &column : columns) {
    column.finalize();

    // PANDAS' default behavior of enabling doublequote for two consecutive
    // quotechars in quoted fields results in reduction to a single quotechar
    if (column->dtype == GDF_STRING &&
        (opts.quotechar != '\0' && opts.doublequote == true)) {
      const std::string quotechar(1, opts.quotechar);
      const std::string dblquotechar(2, opts.quotechar);
      auto str_data = static_cast<NVStrings *>(column->data);
      column->data = str_data->replace(dblquotechar.c_str(), quotechar.c_str());
      NVStrings::destroy(str_data);
    }
  }*/

  return { std::make_unique<table>(std::move(out_columns)), std::move(metadata) };
}

void reader::impl::gather_row_offsets(const char *h_data, size_t h_size,
                                      size_t range_offset,
                                      hipStream_t stream) {
  // Account for the start and end of row region offsets
  const bool require_first_line_start = (range_offset == 0);
  const bool require_last_line_end = (h_data[h_size - 1] != opts.terminator);

  auto symbols = (opts.quotechar != '\0')
                     ? std::vector<char>{opts.terminator, opts.quotechar}
                     : std::vector<char>{opts.terminator};
  const auto num_rows = count_all_from_set(h_data, h_size, symbols) +
                        (require_first_line_start ? 1 : 0);
  const auto num_offsets = num_rows + (require_last_line_end ? 1 : 0);
  row_offsets.resize(num_offsets);

  auto ptr_first = row_offsets.data().get();
  auto ptr_last = ptr_first + num_rows;
  if (require_first_line_start) {
    ptr_first++;
    const uint64_t first_entry = 0;
    row_offsets.front() = first_entry;
  }
  if (require_last_line_end) {
    const uint64_t last_entry = h_size;
    row_offsets.back() = last_entry;
  }

  // Passing offset = 1 to return positions AFTER the found character
  find_all_from_set(h_data, h_size, symbols, 1, ptr_first);

  // Sort the row info according to ascending start offset
  // Subsequent processing (filtering, etc.) may require row order
  thrust::sort(rmm::exec_policy(stream)->on(stream), ptr_first, ptr_last);
}

std::pair<uint64_t, uint64_t> reader::impl::select_rows(
    const char *h_data, size_t h_size, size_t range_size,
    cudf::size_type skip_rows, cudf::size_type skip_end_rows,
    cudf::size_type num_rows, hipStream_t stream) {
  thrust::host_vector<uint64_t> h_row_offsets = row_offsets;
  auto it_begin = h_row_offsets.begin();
  auto it_end = h_row_offsets.end();
  assert(std::distance(it_begin, it_end) >= 1);

  // Currently, ignoring lineterminations within quotes is handled by recording
  // the records of both, and then filtering out the records that is a quotechar
  // or a linetermination within a quotechar pair.
  if (opts.quotechar != '\0') {
    auto count = std::distance(it_begin, it_end) - 1;

    auto filtered_count = count;
    bool quotation = false;
    for (int i = 1; i < count; ++i) {
      if (h_data[h_row_offsets[i] - 1] == opts.quotechar) {
        quotation = !quotation;
        h_row_offsets[i] = static_cast<uint64_t>(-1);
        filtered_count--;
      } else if (quotation) {
        h_row_offsets[i] = static_cast<uint64_t>(-1);
        filtered_count--;
      }
    }
    if (filtered_count != count) {
      it_end = std::remove_if(it_begin, it_end, [](uint64_t pos) {
        return (pos == static_cast<uint64_t>(-1));
      });
    }
  }

  // Exclude the rows that are to be skipped from the start
  if (skip_rows != 0 && skip_rows < std::distance(it_begin, it_end)) {
    it_begin += skip_rows;
  }

  // Exclude the rows outside of requested range
  if (range_size != 0) {
    auto it = it_end - 1;
    while (it >= it_begin && *it > static_cast<uint64_t>(range_size)) {
      --it;
    }
    if ((it + 2) < it_end) {
      it_end = it + 2;
    }
  }

  // Exclude the rows without data
  if (opts.skipblanklines || opts.comment != '\0') {
    const auto newline = opts.skipblanklines ? opts.terminator : opts.comment;
    const auto comment = opts.comment != '\0' ? opts.comment : newline;
    const auto carriage =
        (opts.skipblanklines && opts.terminator == '\n') ? '\r' : comment;

    it_end = std::remove_if(it_begin, it_end, [=, &h_data](uint64_t pos) {
      return ((pos != h_size) &&
              (h_data[pos] == newline || h_data[pos] == comment ||
               h_data[pos] == carriage));
    });
  }

  // Exclude the rows before the header row (inclusive)
  if (std::distance(it_begin, it_end) > 1) {
    if (args_.header == -1) {
      header.assign(h_data + *(it_begin), h_data + *(it_begin + 1));
    } else {
      header.assign(h_data + *(it_begin + args_.header),
                    h_data + *(it_begin + args_.header + 1));
      it_begin += args_.header + 1;
    }
  }

  // Exclude the rows that exceed past the requested number
  if (num_rows >= 0 && num_rows < std::distance(it_begin, it_end)) {
    it_end = it_begin + num_rows + 1;
  }

  // Exclude the rows that are to be skipped from the end
  if (skip_end_rows != 0 && skip_end_rows < std::distance(it_begin, it_end)) {
    it_end -= skip_end_rows;
  }

  const uint64_t offset_start = *it_begin;
  const uint64_t offset_end = *(it_end - 1);

  // Copy out the row starts to use for row-column data parsing
  if (offset_start != offset_end) {
    if (offset_start != 0) {
      for (auto it = it_begin; it != it_end; ++it) {
        *it -= offset_start;
      }
    }
    CUDA_TRY(hipMemcpyAsync(row_offsets.data().get(), &(*it_begin),
                             std::distance(it_begin, it_end) * sizeof(uint64_t),
                             hipMemcpyHostToDevice, stream));

    // Exclude the end-of-data row from number of rows with actual data
    num_records = std::distance(it_begin, it_end) - 1;
  }

  return std::make_pair(offset_start, offset_end);
}

std::vector<data_type> reader::impl::gather_column_types(hipStream_t stream) {
  std::vector<data_type> dtypes;

  if (args_.dtype.empty()) {
    if (num_records == 0) {
      dtypes.resize(num_active_cols, data_type{EMPTY});
    } else {
      d_column_flags = h_column_flags;

      hostdevice_vector<column_parse::stats> column_stats(num_active_cols);
      CUDA_TRY(hipMemsetAsync(column_stats.device_ptr(), 0,
                               column_stats.memory_size(), stream));
      CUDA_TRY(cudf::io::csv::gpu::DetectColumnTypes(
          static_cast<const char *>(data_.data()), row_offsets.data().get(),
          num_records, num_actual_cols, opts, d_column_flags.data().get(),
          column_stats.device_ptr(), stream));
      CUDA_TRY(hipMemcpyAsync(
          column_stats.host_ptr(), column_stats.device_ptr(),
          column_stats.memory_size(), hipMemcpyDeviceToHost, stream));
      CUDA_TRY(hipStreamSynchronize(stream));

      for (int col = 0; col < num_active_cols; col++) {
        unsigned long long countInt =
            column_stats[col].countInt8 + column_stats[col].countInt16 +
            column_stats[col].countInt32 + column_stats[col].countInt64;

        if (column_stats[col].countNULL == num_records) {
          // Entire column is NULL; allocate the smallest amount of memory
          dtypes.emplace_back(cudf::type_id::INT8);
        } else if (column_stats[col].countString > 0L) {
          dtypes.emplace_back(cudf::type_id::STRING);
        } else if (column_stats[col].countDateAndTime > 0L) {
          dtypes.emplace_back(cudf::type_id::TIMESTAMP_NANOSECONDS);
        } else if (column_stats[col].countBool > 0L) {
          dtypes.emplace_back(cudf::type_id::BOOL8);
        } else if (column_stats[col].countFloat > 0L ||
                   (column_stats[col].countFloat == 0L && countInt > 0L &&
                    column_stats[col].countNULL > 0L)) {
          // The second condition has been added to conform to
          // PANDAS which states that a column of integers with
          // a single NULL record need to be treated as floats.
          dtypes.emplace_back(cudf::type_id::FLOAT64);
        } else {
          // All other integers are stored as 64-bit to conform to PANDAS
          dtypes.emplace_back(cudf::type_id::INT64);
        }
      }
    }
  } else {
    const bool is_dict = std::all_of(
        args_.dtype.begin(), args_.dtype.end(),
        [](const auto &s) { return s.find(':') != std::string::npos; });

    if (!is_dict) {
      if (args_.dtype.size() == 1) {
        // If it's a single dtype, assign that dtype to all active columns
        data_type dtype_;
        column_parse::flags col_flags_;
        std::tie(dtype_, col_flags_) = get_dtype_info(args_.dtype[0]);
        dtypes.resize(num_active_cols, dtype_);
        for (int col = 0; col < num_actual_cols; col++) {
          h_column_flags[col] |= col_flags_;
        }
        CUDF_EXPECTS(dtypes.back().id() != cudf::type_id::EMPTY,
                     "Unsupported data type");
      } else {
        // If it's a list, assign dtypes to active columns in the given order
        CUDF_EXPECTS(static_cast<int>(args_.dtype.size()) >= num_actual_cols,
                     "Must specify data types for all columns");

        auto dtype_ = std::back_inserter(dtypes);

        for (int col = 0; col < num_actual_cols; col++) {
          if (h_column_flags[col] & column_parse::enabled) {
            column_parse::flags col_flags_;
            std::tie(dtype_, col_flags_) = get_dtype_info(args_.dtype[col]);
            h_column_flags[col] |= col_flags_;
            CUDF_EXPECTS(dtypes.back().id() != cudf::type_id::EMPTY,
                         "Unsupported data type");
          }
        }
      }
    } else {
      // Translate vector of `name : dtype` strings to map
      // NOTE: Incoming pairs can be out-of-order from column names in dataset
      std::unordered_map<std::string, std::string> col_type_map;
      for (const auto &pair : args_.dtype) {
        const auto pos = pair.find_last_of(':');
        const auto name = pair.substr(0, pos);
        const auto dtype = pair.substr(pos + 1, pair.size());
        col_type_map[name] = dtype;
      }

      auto dtype_ = std::back_inserter(dtypes);

      for (int col = 0; col < num_actual_cols; col++) {
        if (h_column_flags[col] & column_parse::enabled) {
          CUDF_EXPECTS(col_type_map.find(col_names[col]) != col_type_map.end(),
                       "Must specify data types for all active columns");
          column_parse::flags col_flags_;
          std::tie(dtype_, col_flags_) =
              get_dtype_info(col_type_map[col_names[col]]);
          h_column_flags[col] |= col_flags_;
          CUDF_EXPECTS(dtypes.back().id() != cudf::type_id::EMPTY,
                       "Unsupported data type");
        }
      }
    }
  }

  if (args_.timestamp_type.id() != cudf::type_id::EMPTY) {
    for (auto &type : dtypes) {
      if (cudf::is_timestamp(type)) {
        type = args_.timestamp_type;
      }
    }
  }

  return dtypes;
}

void reader::impl::decode_data(const std::vector<data_type> &column_types,
                               std::vector<column_buffer> &out_buffers,
                               hipStream_t stream) {
  thrust::host_vector<void *> h_data(num_active_cols);
  thrust::host_vector<bitmask_type *> h_valid(num_active_cols);

  for (int i = 0; i < num_active_cols; ++i) {
    h_data[i] = out_buffers[i].data();
    h_valid[i] = out_buffers[i].null_mask();
  }

  rmm::device_vector<data_type> d_dtypes(column_types);
  rmm::device_vector<void *> d_data = h_data;
  rmm::device_vector<bitmask_type *> d_valid = h_valid;
  d_column_flags = h_column_flags;

  CUDA_TRY(cudf::io::csv::gpu::DecodeRowColumnData(
      static_cast<const char *>(data_.data()), row_offsets.data().get(),
      num_records, num_actual_cols, opts, d_column_flags.data().get(),
      d_dtypes.data().get(), d_data.data().get(), d_valid.data().get(),
      stream));
  CUDA_TRY(hipStreamSynchronize(stream));

  for (int i = 0; i < num_active_cols; ++i) {
    out_buffers[i].null_count() = UNKNOWN_NULL_COUNT;
  }
}

reader::impl::impl(std::unique_ptr<datasource> source, std::string filepath,
                   reader_options const &options,
                   rmm::mr::device_memory_resource *mr)
    : source_(std::move(source)), mr_(mr), filepath_(filepath), args_(options) {
  num_actual_cols = args_.names.size();
  num_active_cols = args_.names.size();

  if (args_.delim_whitespace) {
    opts.delimiter = ' ';
    opts.multi_delimiter = true;
  } else {
    opts.delimiter = args_.delimiter;
    opts.multi_delimiter = false;
  }
  opts.terminator = args_.lineterminator;
  if (args_.quotechar != '\0' && args_.quoting != quote_style::NONE) {
    opts.quotechar = args_.quotechar;
    opts.keepquotes = false;
    opts.doublequote = args_.doublequote;
  } else {
    opts.quotechar = '\0';
    opts.keepquotes = true;
    opts.doublequote = false;
  }
  opts.skipblanklines = args_.skip_blank_lines;
  opts.comment = args_.comment;
  opts.dayfirst = args_.dayfirst;
  opts.decimal = args_.decimal;
  opts.thousands = args_.thousands;
  CUDF_EXPECTS(opts.decimal != opts.delimiter,
               "Decimal point cannot be the same as the delimiter");
  CUDF_EXPECTS(opts.thousands != opts.delimiter,
               "Thousands separator cannot be the same as the delimiter");

  compression_type_ = infer_compression_type(
      args_.compression, filepath,
      {{"gz", "gzip"}, {"zip", "zip"}, {"bz2", "bz2"}, {"xz", "xz"}});

  // Handle user-defined false values, whereby field data is substituted with a
  // boolean true or numeric `1` value
  if (args_.true_values.size() != 0) {
    d_trueTrie = createSerializedTrie(args_.true_values);
    opts.trueValuesTrie = d_trueTrie.data().get();
  }

  // Handle user-defined false values, whereby field data is substituted with a
  // boolean false or numeric `0` value
  if (args_.false_values.size() != 0) {
    d_falseTrie = createSerializedTrie(args_.false_values);
    opts.falseValuesTrie = d_falseTrie.data().get();
  }

  // Handle user-defined N/A values, whereby field data is treated as null
  if (args_.na_values.size() != 0) {
    d_naTrie = createSerializedTrie(args_.na_values);
    opts.naValuesTrie = d_naTrie.data().get();
  }
}

// Forward to implementation
reader::reader(std::string filepath, reader_options const &options,
               rmm::mr::device_memory_resource *mr)
    : _impl(std::make_unique<impl>(nullptr, filepath, options, mr)) {
  // Delay actual instantiation of data source until read to allow for
  // partial memory mapping of file using byte ranges
}

// Forward to implementation
reader::reader(const char *buffer, size_t length, reader_options const &options,
               rmm::mr::device_memory_resource *mr)
    : _impl(std::make_unique<impl>(datasource::create(buffer, length), "",
                                   options, mr)) {}

// Forward to implementation
reader::reader(std::shared_ptr<arrow::io::RandomAccessFile> file,
               reader_options const &options,
               rmm::mr::device_memory_resource *mr)
    : _impl(std::make_unique<impl>(datasource::create(file), "", options, mr)) {
}

// Destructor within this translation unit
reader::~reader() = default;

// Forward to implementation
table_with_metadata reader::read_all(hipStream_t stream) {
  return _impl->read(0, 0, 0, 0, -1, stream);
}

// Forward to implementation
table_with_metadata reader::read_byte_range(size_t offset, size_t size,
                                            hipStream_t stream) {
  return _impl->read(offset, size, 0, 0, -1, stream);
}

// Forward to implementation
table_with_metadata reader::read_rows(size_type num_skip_header,
                                      size_type num_skip_footer,
                                      size_type num_rows,
                                      hipStream_t stream) {
  CUDF_EXPECTS(num_rows == -1 || num_skip_footer == 0,
               "Cannot use both `num_rows` and `num_skip_footer`");

  return _impl->read(0, 0, num_skip_header, num_skip_footer, num_rows, stream);
}

}  // namespace csv
}  // namespace detail
}  // namespace io
}  // namespace experimental
}  // namespace cudf
