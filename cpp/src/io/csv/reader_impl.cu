#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019-2020, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

/**
 * @file reader_impl.cu
 * @brief cuDF-IO CSV reader class implementation
 **/

#include "reader_impl.hpp"

#include <io/comp/io_uncomp.h>
#include <io/utilities/parsing_utils.cuh>
#include <io/utilities/type_conversion.cuh>

#include <cudf/io/types.hpp>
#include <cudf/strings/replace.hpp>
#include <cudf/table/table.hpp>
#include <cudf/utilities/error.hpp>
#include <cudf/utilities/span.hpp>

#include <rmm/cuda_stream_view.hpp>

#include <algorithm>
#include <iostream>
#include <numeric>
#include <tuple>
#include <unordered_map>

using std::string;
using std::vector;

using cudf::detail::device_span;
using cudf::detail::host_span;

namespace cudf {
namespace io {
namespace detail {
namespace csv {
using namespace cudf::io::csv;
using namespace cudf::io;

/**
 * @brief Estimates the maximum expected length or a row, based on the number
 * of columns
 *
 * If the number of columns is not available, it will return a value large
 * enough for most use cases
 *
 * @param[in] num_columns Number of columns in the CSV file (optional)
 *
 * @return Estimated maximum size of a row, in bytes
 **/
constexpr size_t calculateMaxRowSize(int num_columns = 0) noexcept
{
  constexpr size_t max_row_bytes = 16 * 1024;  // 16KB
  constexpr size_t column_bytes  = 64;
  constexpr size_t base_padding  = 1024;  // 1KB
  if (num_columns == 0) {
    // Use flat size if the number of columns is not known
    return max_row_bytes;
  } else {
    // Expand the size based on the number of columns, if available
    return base_padding + num_columns * column_bytes;
  }
}

/**
 * @brief Translates a dtype string and returns its dtype enumeration and any
 * extended dtype flags that are supported by cuIO. Often, this is a column
 * with the same underlying dtype the basic types, but with different parsing
 * interpretations.
 *
 * @param[in] dtype String containing the basic or extended dtype
 *
 * @return Tuple of data_type and flags
 */
std::tuple<data_type, column_parse::flags> get_dtype_info(const std::string &dtype)
{
  if (dtype == "hex" || dtype == "hex64") {
    return std::make_tuple(data_type{cudf::type_id::INT64}, column_parse::as_hexadecimal);
  }
  if (dtype == "hex32") {
    return std::make_tuple(data_type{cudf::type_id::INT32}, column_parse::as_hexadecimal);
  }

  return std::make_tuple(convert_string_to_dtype(dtype), column_parse::as_default);
}

/**
 * @brief Removes the first and Last quote in the string
 */
string removeQuotes(string str, char quotechar)
{
  // Exclude first and last quotation char
  const size_t first_quote = str.find(quotechar);
  if (first_quote != string::npos) { str.erase(first_quote, 1); }
  const size_t last_quote = str.rfind(quotechar);
  if (last_quote != string::npos) { str.erase(last_quote, 1); }

  return str;
}

/**
 * @brief Parse the first row to set the column names in the raw_csv parameter.
 * The first row can be either the header row, or the first data row
 */
std::vector<std::string> setColumnNames(std::vector<char> const &header,
                                        parse_options_view const &opts,
                                        int header_row,
                                        std::string prefix)
{
  std::vector<std::string> col_names;

  // If there is only a single character then it would be the terminator
  if (header.size() <= 1) { return col_names; }

  std::vector<char> first_row = header;
  int num_cols                = 0;

  bool quotation = false;
  for (size_t pos = 0, prev = 0; pos < first_row.size(); ++pos) {
    // Flip the quotation flag if current character is a quotechar
    if (first_row[pos] == opts.quotechar) {
      quotation = !quotation;
    }
    // Check if end of a column/row
    else if (pos == first_row.size() - 1 || (!quotation && first_row[pos] == opts.terminator) ||
             (!quotation && first_row[pos] == opts.delimiter)) {
      // This is the header, add the column name
      if (header_row >= 0) {
        // Include the current character, in case the line is not terminated
        int col_name_len = pos - prev + 1;
        // Exclude the delimiter/terminator is present
        if (first_row[pos] == opts.delimiter || first_row[pos] == opts.terminator) {
          --col_name_len;
        }
        // Also exclude '\r' character at the end of the column name if it's
        // part of the terminator
        if (col_name_len > 0 && opts.terminator == '\n' && first_row[pos] == '\n' &&
            first_row[pos - 1] == '\r') {
          --col_name_len;
        }

        const string new_col_name(first_row.data() + prev, col_name_len);
        col_names.push_back(removeQuotes(new_col_name, opts.quotechar));

        // Stop parsing when we hit the line terminator; relevant when there is
        // a blank line following the header. In this case, first_row includes
        // multiple line terminators at the end, as the new recStart belongs to
        // a line that comes after the blank line(s)
        if (!quotation && first_row[pos] == opts.terminator) { break; }
      } else {
        // This is the first data row, add the automatically generated name
        col_names.push_back(prefix + std::to_string(num_cols));
      }
      num_cols++;

      // Skip adjacent delimiters if delim_whitespace is set
      while (opts.multi_delimiter && pos < first_row.size() && first_row[pos] == opts.delimiter &&
             first_row[pos + 1] == opts.delimiter) {
        ++pos;
      }
      prev = pos + 1;
    }
  }

  return col_names;
}

table_with_metadata reader::impl::read(rmm::cuda_stream_view stream)
{
  auto range_offset  = opts_.get_byte_range_offset();
  auto range_size    = opts_.get_byte_range_size();
  auto skip_rows     = opts_.get_skiprows();
  auto skip_end_rows = opts_.get_skipfooter();
  auto num_rows      = opts_.get_nrows();

  if (range_offset > 0 || range_size > 0) {
    CUDF_EXPECTS(compression_type_ == "none",
                 "Reading compressed data using `byte range` is unsupported");
  }
  size_t map_range_size = 0;
  if (range_size != 0) {
    const auto num_columns = std::max(opts_.get_names().size(), opts_.get_dtypes().size());
    map_range_size         = range_size + calculateMaxRowSize(num_columns);
  }

  // Support delayed opening of the file if using memory mapping datasource
  // This allows only mapping of a subset of the file if using byte range
  if (source_ == nullptr) {
    assert(!filepath_.empty());
    source_ = datasource::create(filepath_, range_offset, map_range_size);
  }

  // Return an empty dataframe if no data and no column metadata to process
  if (source_->is_empty() && (opts_.get_names().empty() || opts_.get_dtypes().empty())) {
    return {std::make_unique<table>(), {}};
  }

  // Transfer source data to GPU
  if (!source_->is_empty()) {
    auto data_size = (map_range_size != 0) ? map_range_size : source_->size();
    auto buffer    = source_->host_read(range_offset, data_size);

    auto h_data = host_span<char const>(  //
      reinterpret_cast<const char *>(buffer->data()),
      buffer->size());

    std::vector<char> h_uncomp_data_owner;

    if (compression_type_ != "none") {
      h_uncomp_data_owner = get_uncompressed_data(h_data, compression_type_);
      h_data              = h_uncomp_data_owner;
    }
    // None of the parameters for row selection is used, we are parsing the entire file
    const bool load_whole_file = range_offset == 0 && range_size == 0 && skip_rows <= 0 &&
                                 skip_end_rows <= 0 && num_rows == -1;

    // With byte range, find the start of the first data row
    size_t const data_start_offset = (range_offset != 0) ? find_first_row_start(h_data) : 0;

    // TODO: Allow parsing the header outside the mapped range
    CUDF_EXPECTS((range_offset == 0 || opts_.get_header() < 0),
                 "byte_range offset with header not supported");

    // Gather row offsets
    gather_row_offsets(h_data,
                       data_start_offset,
                       (range_size) ? range_size : h_data.size(),
                       (skip_rows > 0) ? skip_rows : 0,
                       num_rows,
                       load_whole_file,
                       stream);

    // Exclude the rows that are to be skipped from the end
    if (skip_end_rows > 0 && static_cast<size_t>(skip_end_rows) < row_offsets_.size()) {
      row_offsets_.resize(row_offsets_.size() - skip_end_rows);
    }

    // Exclude the end-of-data row from number of rows with actual data
    num_records_ = row_offsets_.size();
    num_records_ -= (num_records_ > 0);
  } else {
    num_records_ = 0;
  }

  // Check if the user gave us a list of column names
  if (not opts_.get_names().empty()) {
    h_column_flags_.resize(opts_.get_names().size(), column_parse::enabled);
    col_names_ = opts_.get_names();
  } else {
    col_names_ = setColumnNames(header_, opts.view(), opts_.get_header(), opts_.get_prefix());

    num_actual_cols_ = num_active_cols_ = col_names_.size();

    h_column_flags_.resize(num_actual_cols_, column_parse::enabled);

    // Rename empty column names to "Unnamed: col_index"
    for (size_t col_idx = 0; col_idx < col_names_.size(); ++col_idx) {
      if (col_names_[col_idx].empty()) {
        col_names_[col_idx] = string("Unnamed: ") + std::to_string(col_idx);
      }
    }

    // Looking for duplicates
    std::unordered_map<string, int> col_names_histogram;
    for (auto &col_name : col_names_) {
      // Operator [] inserts a default-initialized value if the given key is not
      // present
      if (++col_names_histogram[col_name] > 1) {
        if (opts_.is_enabled_mangle_dupe_cols()) {
          // Rename duplicates of column X as X.1, X.2, ...; First appearance
          // stays as X
          col_name += "." + std::to_string(col_names_histogram[col_name] - 1);
        } else {
          // All duplicate columns will be ignored; First appearance is parsed
          const auto idx       = &col_name - col_names_.data();
          h_column_flags_[idx] = column_parse::disabled;
        }
      }
    }

    // Update the number of columns to be processed, if some might have been
    // removed
    if (!opts_.is_enabled_mangle_dupe_cols()) { num_active_cols_ = col_names_histogram.size(); }
  }

  // User can specify which columns should be parsed
  if (!opts_.get_use_cols_indexes().empty() || !opts_.get_use_cols_names().empty()) {
    std::fill(h_column_flags_.begin(), h_column_flags_.end(), column_parse::disabled);

    for (const auto index : opts_.get_use_cols_indexes()) {
      h_column_flags_[index] = column_parse::enabled;
    }
    num_active_cols_ = opts_.get_use_cols_indexes().size();

    for (const auto &name : opts_.get_use_cols_names()) {
      const auto it = std::find(col_names_.begin(), col_names_.end(), name);
      if (it != col_names_.end()) {
        h_column_flags_[it - col_names_.begin()] = column_parse::enabled;
        num_active_cols_++;
      }
    }
  }

  // User can specify which columns should be inferred as datetime
  if (!opts_.get_infer_date_indexes().empty() || !opts_.get_infer_date_names().empty()) {
    for (const auto index : opts_.get_infer_date_indexes()) {
      h_column_flags_[index] |= column_parse::as_datetime;
    }

    for (const auto &name : opts_.get_infer_date_names()) {
      auto it = std::find(col_names_.begin(), col_names_.end(), name);
      if (it != col_names_.end()) {
        h_column_flags_[it - col_names_.begin()] |= column_parse::as_datetime;
      }
    }
  }

  // Return empty table rather than exception if nothing to load
  if (num_active_cols_ == 0) { return {std::make_unique<table>(), {}}; }

  auto metadata     = table_metadata{};
  auto out_columns  = std::vector<std::unique_ptr<cudf::column>>();
  auto column_types = gather_column_types(stream);

  out_columns.reserve(column_types.size());

  if (num_records_ != 0) {
    auto out_buffers = decode_data(column_types, stream);
    for (size_t i = 0; i < column_types.size(); ++i) {
      metadata.column_names.emplace_back(out_buffers[i].name);
      if (column_types[i].id() == type_id::STRING && opts.quotechar != '\0' &&
          opts.doublequote == true) {
        // PANDAS' default behavior of enabling doublequote for two consecutive
        // quotechars in quoted fields results in reduction to a single quotechar
        // TODO: Would be much more efficient to perform this operation in-place
        // during the conversion stage
        const std::string quotechar(1, opts.quotechar);
        const std::string dblquotechar(2, opts.quotechar);
        std::unique_ptr<column> col = make_strings_column(out_buffers[i]._strings, stream);
        out_columns.emplace_back(
          cudf::strings::replace(col->view(), dblquotechar, quotechar, -1, mr_));
      } else {
        out_columns.emplace_back(make_column(out_buffers[i], nullptr, stream, mr_));
      }
    }
  } else {
    // Create empty columns
    for (size_t i = 0; i < column_types.size(); ++i) {
      out_columns.emplace_back(make_empty_column(column_types[i]));
    }
    // Handle empty metadata
    for (int col = 0; col < num_actual_cols_; ++col) {
      if (h_column_flags_[col] & column_parse::enabled) {
        metadata.column_names.emplace_back(col_names_[col]);
      }
    }
  }
  return {std::make_unique<table>(std::move(out_columns)), std::move(metadata)};
}

size_t reader::impl::find_first_row_start(host_span<char const> const data)
{
  // For now, look for the first terminator (assume the first terminator isn't within a quote)
  // TODO: Attempt to infer this from the data
  size_t pos = 0;
  while (pos < data.size() && data[pos] != opts.terminator) { ++pos; }
  return std::min(pos + 1, data.size());
}

void reader::impl::gather_row_offsets(host_span<char const> const data,
                                      size_t range_begin,
                                      size_t range_end,
                                      size_t skip_rows,
                                      int64_t num_rows,
                                      bool load_whole_file,
                                      rmm::cuda_stream_view stream)
{
  constexpr size_t max_chunk_bytes = 64 * 1024 * 1024;  // 64MB
  size_t buffer_size               = std::min(max_chunk_bytes, data.size());
  size_t max_blocks =
    std::max<size_t>((buffer_size / cudf::io::csv::gpu::rowofs_block_bytes) + 1, 2);
  hostdevice_vector<uint64_t> row_ctx(max_blocks);
  size_t buffer_pos  = std::min(range_begin - std::min(range_begin, sizeof(char)), data.size());
  size_t pos         = std::min(range_begin, data.size());
  size_t header_rows = (opts_.get_header() >= 0) ? opts_.get_header() + 1 : 0;
  uint64_t ctx       = 0;

  // For compatibility with the previous parser, a row is considered in-range if the
  // previous row terminator is within the given range
  range_end += (range_end < data.size());
  data_.resize(0);
  row_offsets_.resize(0);
  data_.reserve((load_whole_file) ? data.size() : std::min(buffer_size * 2, data.size()));
  do {
    size_t target_pos = std::min(pos + max_chunk_bytes, data.size());
    size_t chunk_size = target_pos - pos;

    data_.insert(data_.end(), data.begin() + buffer_pos + data_.size(), data.begin() + target_pos);

    // Pass 1: Count the potential number of rows in each character block for each
    // possible parser state at the beginning of the block.
    uint32_t num_blocks = cudf::io::csv::gpu::gather_row_offsets(opts.view(),
                                                                 row_ctx.device_ptr(),
                                                                 device_span<uint64_t>(),
                                                                 data_,
                                                                 chunk_size,
                                                                 pos,
                                                                 buffer_pos,
                                                                 data.size(),
                                                                 range_begin,
                                                                 range_end,
                                                                 skip_rows,
                                                                 stream);
    CUDA_TRY(hipMemcpyAsync(row_ctx.host_ptr(),
                             row_ctx.device_ptr(),
                             num_blocks * sizeof(uint64_t),
                             hipMemcpyDeviceToHost,
                             stream.value()));
    stream.synchronize();

    // Sum up the rows in each character block, selecting the row count that
    // corresponds to the current input context. Also stores the now known input
    // context per character block that will be needed by the second pass.
    for (uint32_t i = 0; i < num_blocks; i++) {
      uint64_t ctx_next = cudf::io::csv::gpu::select_row_context(ctx, row_ctx[i]);
      row_ctx[i]        = ctx;
      ctx               = ctx_next;
    }
    size_t total_rows = ctx >> 2;
    if (total_rows > skip_rows) {
      // At least one row in range in this batch
      row_offsets_.resize(total_rows - skip_rows);

      CUDA_TRY(hipMemcpyAsync(row_ctx.device_ptr(),
                               row_ctx.host_ptr(),
                               num_blocks * sizeof(uint64_t),
                               hipMemcpyHostToDevice,
                               stream.value()));

      // Pass 2: Output row offsets
      cudf::io::csv::gpu::gather_row_offsets(opts.view(),
                                             row_ctx.device_ptr(),
                                             row_offsets_,
                                             data_,
                                             chunk_size,
                                             pos,
                                             buffer_pos,
                                             data.size(),
                                             range_begin,
                                             range_end,
                                             skip_rows,
                                             stream);
      // With byte range, we want to keep only one row out of the specified range
      if (range_end < data.size()) {
        CUDA_TRY(hipMemcpyAsync(row_ctx.host_ptr(),
                                 row_ctx.device_ptr(),
                                 num_blocks * sizeof(uint64_t),
                                 hipMemcpyDeviceToHost,
                                 stream.value()));
        stream.synchronize();

        size_t rows_out_of_range = 0;
        for (uint32_t i = 0; i < num_blocks; i++) { rows_out_of_range += row_ctx[i]; }
        if (rows_out_of_range != 0) {
          // Keep one row out of range (used to infer length of previous row)
          auto new_row_offsets_size =
            row_offsets_.size() - std::min(rows_out_of_range - 1, row_offsets_.size());
          row_offsets_.resize(new_row_offsets_size);
          // Implies we reached the end of the range
          break;
        }
      }
      // num_rows does not include blank rows
      if (num_rows >= 0) {
        if (row_offsets_.size() > header_rows + static_cast<size_t>(num_rows)) {
          size_t num_blanks =
            cudf::io::csv::gpu::count_blank_rows(opts.view(), data_, row_offsets_, stream);
          if (row_offsets_.size() - num_blanks > header_rows + static_cast<size_t>(num_rows)) {
            // Got the desired number of rows
            break;
          }
        }
      }
    } else {
      // Discard data (all rows below skip_rows), keeping one character for history
      size_t discard_bytes = std::max(data_.size(), sizeof(char)) - sizeof(char);
      if (discard_bytes != 0) {
        data_.erase(data_.begin(), data_.begin() + discard_bytes);
        buffer_pos += discard_bytes;
      }
    }
    pos = target_pos;
  } while (pos < data.size());

  // Eliminate blank rows
  if (row_offsets_.size() != 0) {
    cudf::io::csv::gpu::remove_blank_rows(opts.view(), data_, row_offsets_, stream);
  }
  // Remove header rows and extract header
  const size_t header_row_index = std::max<size_t>(header_rows, 1) - 1;
  if (header_row_index + 1 < row_offsets_.size()) {
    CUDA_TRY(hipMemcpyAsync(row_ctx.host_ptr(),
                             row_offsets_.data().get() + header_row_index,
                             2 * sizeof(uint64_t),
                             hipMemcpyDeviceToHost,
                             stream.value()));
    stream.synchronize();

    const auto header_start = buffer_pos + row_ctx[0];
    const auto header_end   = buffer_pos + row_ctx[1];
    CUDF_EXPECTS(header_start <= header_end && header_end <= data.size(),
                 "Invalid csv header location");
    header_.assign(data.begin() + header_start, data.begin() + header_end);
    if (header_rows > 0) {
      row_offsets_.erase(row_offsets_.begin(), row_offsets_.begin() + header_rows);
    }
  }
  // Apply num_rows limit
  if (num_rows >= 0) { row_offsets_.resize(std::min<size_t>(row_offsets_.size(), num_rows + 1)); }
}

std::vector<data_type> reader::impl::gather_column_types(rmm::cuda_stream_view stream)
{
  std::vector<data_type> dtypes;

  if (opts_.get_dtypes().empty()) {
    if (num_records_ == 0) {
      dtypes.resize(num_active_cols_, data_type{type_id::EMPTY});
    } else {
      d_column_flags_ = h_column_flags_;

      auto column_stats = cudf::io::csv::gpu::detect_column_types(
        opts.view(), data_, d_column_flags_, row_offsets_, num_active_cols_, stream);

      stream.synchronize();

      for (int col = 0; col < num_active_cols_; col++) {
        unsigned long long int_count_total = column_stats[col].big_int_count +
                                             column_stats[col].negative_small_int_count +
                                             column_stats[col].positive_small_int_count;

        if (column_stats[col].null_count == num_records_) {
          // Entire column is NULL; allocate the smallest amount of memory
          dtypes.emplace_back(cudf::type_id::INT8);
        } else if (column_stats[col].string_count > 0L) {
          dtypes.emplace_back(cudf::type_id::STRING);
        } else if (column_stats[col].datetime_count > 0L) {
          dtypes.emplace_back(cudf::type_id::TIMESTAMP_NANOSECONDS);
        } else if (column_stats[col].bool_count > 0L) {
          dtypes.emplace_back(cudf::type_id::BOOL8);
        } else if (column_stats[col].float_count > 0L ||
                   (column_stats[col].float_count == 0L && int_count_total > 0L &&
                    column_stats[col].null_count > 0L)) {
          // The second condition has been added to conform to
          // PANDAS which states that a column of integers with
          // a single NULL record need to be treated as floats.
          dtypes.emplace_back(cudf::type_id::FLOAT64);
        } else if (column_stats[col].big_int_count == 0) {
          dtypes.emplace_back(cudf::type_id::INT64);
        } else if (column_stats[col].big_int_count != 0 &&
                   column_stats[col].negative_small_int_count != 0) {
          dtypes.emplace_back(cudf::type_id::STRING);
        } else {
          // Integers are stored as 64-bit to conform to PANDAS
          dtypes.emplace_back(cudf::type_id::UINT64);
        }
      }
    }
  } else {
    const bool is_dict =
      std::all_of(opts_.get_dtypes().begin(), opts_.get_dtypes().end(), [](const auto &s) {
        return s.find(':') != std::string::npos;
      });

    if (!is_dict) {
      if (opts_.get_dtypes().size() == 1) {
        // If it's a single dtype, assign that dtype to all active columns
        data_type dtype_;
        column_parse::flags col_flags_;
        std::tie(dtype_, col_flags_) = get_dtype_info(opts_.get_dtypes()[0]);
        dtypes.resize(num_active_cols_, dtype_);
        for (int col = 0; col < num_actual_cols_; col++) { h_column_flags_[col] |= col_flags_; }
        CUDF_EXPECTS(dtypes.back().id() != cudf::type_id::EMPTY, "Unsupported data type");
      } else {
        // If it's a list, assign dtypes to active columns in the given order
        CUDF_EXPECTS(static_cast<int>(opts_.get_dtypes().size()) >= num_actual_cols_,
                     "Must specify data types for all columns");

        auto dtype_ = std::back_inserter(dtypes);

        for (int col = 0; col < num_actual_cols_; col++) {
          if (h_column_flags_[col] & column_parse::enabled) {
            column_parse::flags col_flags_;
            std::tie(dtype_, col_flags_) = get_dtype_info(opts_.get_dtypes()[col]);
            h_column_flags_[col] |= col_flags_;
            CUDF_EXPECTS(dtypes.back().id() != cudf::type_id::EMPTY, "Unsupported data type");
          }
        }
      }
    } else {
      // Translate vector of `name : dtype` strings to map
      // NOTE: Incoming pairs can be out-of-order from column names in dataset
      std::unordered_map<std::string, std::string> col_type_map;
      for (const auto &pair : opts_.get_dtypes()) {
        const auto pos     = pair.find_last_of(':');
        const auto name    = pair.substr(0, pos);
        const auto dtype   = pair.substr(pos + 1, pair.size());
        col_type_map[name] = dtype;
      }

      auto dtype_ = std::back_inserter(dtypes);

      for (int col = 0; col < num_actual_cols_; col++) {
        if (h_column_flags_[col] & column_parse::enabled) {
          CUDF_EXPECTS(col_type_map.find(col_names_[col]) != col_type_map.end(),
                       "Must specify data types for all active columns");
          column_parse::flags col_flags_;
          std::tie(dtype_, col_flags_) = get_dtype_info(col_type_map[col_names_[col]]);
          h_column_flags_[col] |= col_flags_;
          CUDF_EXPECTS(dtypes.back().id() != cudf::type_id::EMPTY, "Unsupported data type");
        }
      }
    }
  }

  if (opts_.get_timestamp_type().id() != cudf::type_id::EMPTY) {
    for (auto &type : dtypes) {
      if (cudf::is_timestamp(type)) { type = opts_.get_timestamp_type(); }
    }
  }

  for (size_t i = 0; i < dtypes.size(); i++) {
    // Replace EMPTY dtype with STRING
    if (dtypes[i].id() == type_id::EMPTY) { dtypes[i] = data_type{type_id::STRING}; }
  }

  return dtypes;
}

std::vector<column_buffer> reader::impl::decode_data(std::vector<data_type> const &column_types,
                                                     rmm::cuda_stream_view stream)
{
  // Alloc output; columns' data memory is still expected for empty dataframe
  std::vector<column_buffer> out_buffers;

  out_buffers.reserve(column_types.size());

  for (int col = 0, active_col = 0; col < num_actual_cols_; ++col) {
    if (h_column_flags_[col] & column_parse::enabled) {
      const bool is_final_allocation = column_types[active_col].id() != type_id::STRING;
      auto out_buffer =
        column_buffer(column_types[active_col],
                      num_records_,
                      true,
                      stream,
                      is_final_allocation ? mr_ : rmm::mr::get_current_device_resource());

      out_buffer.name = col_names_[col];
      out_buffers.emplace_back(std::move(out_buffer));
      active_col++;
    }
  }

  thrust::host_vector<void *> h_data(num_active_cols_);
  thrust::host_vector<bitmask_type *> h_valid(num_active_cols_);

  for (int i = 0; i < num_active_cols_; ++i) {
    h_data[i]  = out_buffers[i].data();
    h_valid[i] = out_buffers[i].null_mask();
  }

  rmm::device_vector<data_type> d_dtypes(column_types);
  rmm::device_vector<void *> d_data          = h_data;
  rmm::device_vector<bitmask_type *> d_valid = h_valid;
  d_column_flags_                            = h_column_flags_;

  cudf::io::csv::gpu::decode_row_column_data(
    opts.view(), data_, d_column_flags_, row_offsets_, d_dtypes, d_data, d_valid, stream);

  stream.synchronize();

  for (int i = 0; i < num_active_cols_; ++i) { out_buffers[i].null_count() = UNKNOWN_NULL_COUNT; }

  return out_buffers;
}

/**
 * @brief Create a serialized trie for N/A value matching, based on the options.
 */
thrust::host_vector<SerialTrieNode> create_na_trie(char quotechar,
                                                   csv_reader_options const &reader_opts)
{
  // Default values to recognize as null values
  static std::vector<std::string> const default_na_values{"",
                                                          "#N/A",
                                                          "#N/A N/A",
                                                          "#NA",
                                                          "-1.#IND",
                                                          "-1.#QNAN",
                                                          "-NaN",
                                                          "-nan",
                                                          "1.#IND",
                                                          "1.#QNAN",
                                                          "<NA>",
                                                          "N/A",
                                                          "NA",
                                                          "NULL",
                                                          "NaN",
                                                          "n/a",
                                                          "nan",
                                                          "null"};

  if (!reader_opts.is_enabled_na_filter()) { return {}; }

  std::vector<std::string> na_values = reader_opts.get_na_values();
  if (reader_opts.is_enabled_keep_default_na()) {
    na_values.insert(na_values.end(), default_na_values.begin(), default_na_values.end());
  }

  // Pandas treats empty strings as N/A if empty fields are treated as N/A
  if (std::find(na_values.begin(), na_values.end(), "") != na_values.end()) {
    na_values.push_back(std::string(2, quotechar));
  }

  return createSerializedTrie(na_values);
}

parse_options make_parse_options(csv_reader_options const &reader_opts)
{
  auto parse_opts = parse_options{};

  if (reader_opts.is_enabled_delim_whitespace()) {
    parse_opts.delimiter       = ' ';
    parse_opts.multi_delimiter = true;
  } else {
    parse_opts.delimiter       = reader_opts.get_delimiter();
    parse_opts.multi_delimiter = false;
  }

  parse_opts.terminator = reader_opts.get_lineterminator();

  if (reader_opts.get_quotechar() != '\0' && reader_opts.get_quoting() != quote_style::NONE) {
    parse_opts.quotechar   = reader_opts.get_quotechar();
    parse_opts.keepquotes  = false;
    parse_opts.doublequote = reader_opts.is_enabled_doublequote();
  } else {
    parse_opts.quotechar   = '\0';
    parse_opts.keepquotes  = true;
    parse_opts.doublequote = false;
  }

  parse_opts.skipblanklines = reader_opts.is_enabled_skip_blank_lines();
  parse_opts.comment        = reader_opts.get_comment();
  parse_opts.dayfirst       = reader_opts.is_enabled_dayfirst();
  parse_opts.decimal        = reader_opts.get_decimal();
  parse_opts.thousands      = reader_opts.get_thousands();

  CUDF_EXPECTS(parse_opts.decimal != parse_opts.delimiter,
               "Decimal point cannot be the same as the delimiter");
  CUDF_EXPECTS(parse_opts.thousands != parse_opts.delimiter,
               "Thousands separator cannot be the same as the delimiter");

  // Handle user-defined false values, whereby field data is substituted with a
  // boolean true or numeric `1` value
  if (reader_opts.get_true_values().size() != 0) {
    parse_opts.trie_true = createSerializedTrie(reader_opts.get_true_values());
  }

  // Handle user-defined false values, whereby field data is substituted with a
  // boolean false or numeric `0` value
  if (reader_opts.get_false_values().size() != 0) {
    parse_opts.trie_false = createSerializedTrie(reader_opts.get_false_values());
  }

  // Handle user-defined N/A values, whereby field data is treated as null
  parse_opts.trie_na = create_na_trie(parse_opts.quotechar, reader_opts);

  return parse_opts;
}

reader::impl::impl(std::unique_ptr<datasource> source,
                   std::string filepath,
                   csv_reader_options const &options,
                   rmm::mr::device_memory_resource *mr)
  : mr_(mr), source_(std::move(source)), filepath_(filepath), opts_(options)
{
  num_actual_cols_ = opts_.get_names().size();
  num_active_cols_ = num_actual_cols_;

  compression_type_ =
    infer_compression_type(opts_.get_compression(),
                           filepath,
                           {{"gz", "gzip"}, {"zip", "zip"}, {"bz2", "bz2"}, {"xz", "xz"}});

  opts = make_parse_options(options);
}

// Forward to implementation
reader::reader(std::vector<std::string> const &filepaths,
               csv_reader_options const &options,
               rmm::mr::device_memory_resource *mr)
{
  CUDF_EXPECTS(filepaths.size() == 1, "Only a single source is currently supported.");
  // Delay actual instantiation of data source until read to allow for
  // partial memory mapping of file using byte ranges
  _impl = std::make_unique<impl>(nullptr, filepaths[0], options, mr);
}

// Forward to implementation
reader::reader(std::vector<std::unique_ptr<cudf::io::datasource>> &&sources,
               csv_reader_options const &options,
               rmm::mr::device_memory_resource *mr)
{
  CUDF_EXPECTS(sources.size() == 1, "Only a single source is currently supported.");
  _impl = std::make_unique<impl>(std::move(sources[0]), "", options, mr);
}

// Destructor within this translation unit
reader::~reader() = default;

// Forward to implementation
table_with_metadata reader::read(rmm::cuda_stream_view stream) { return _impl->read(stream); }

}  // namespace csv
}  // namespace detail
}  // namespace io
}  // namespace cudf
