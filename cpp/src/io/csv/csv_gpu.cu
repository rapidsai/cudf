#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019-2020, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "csv_common.h"
#include "csv_gpu.h"
#include "datetime.cuh"

#include <io/utilities/block_utils.cuh>
#include <io/utilities/parsing_utils.cuh>

#include <cudf/detail/utilities/trie.cuh>
#include <cudf/fixed_point/fixed_point.hpp>
#include <cudf/lists/list_view.cuh>
#include <cudf/null_mask.hpp>
#include <cudf/strings/string_view.cuh>
#include <cudf/utilities/bit.hpp>
#include <cudf/utilities/error.hpp>
#include <cudf/utilities/span.hpp>
#include <cudf/utilities/traits.hpp>
#include <cudf/utilities/type_dispatcher.hpp>

#include <thrust/detail/copy.h>
#include <thrust/transform.h>

#include <type_traits>

using namespace ::cudf::io;

using cudf::detail::device_span;

namespace cudf {
namespace io {
namespace csv {
namespace gpu {

/// Block dimension for dtype detection and conversion kernels
constexpr uint32_t csvparse_block_dim = 128;

/*
 * @brief Checks whether the given character is a whitespace character.
 *
 * @param ch The character to check
 *
 * @return True if the input is whitespace, False otherwise
 */
__device__ __inline__ bool is_whitespace(char c) { return c == '\t' || c == ' '; }

/*
 * @brief Scans a character stream within a range, and adjusts the start and end
 * indices of the range to ignore whitespace and quotation characters.
 *
 * @param data The character stream to scan
 * @param start The start index to adjust
 * @param end The end index to adjust
 * @param quotechar The character used to denote quotes
 *
 * @return Adjusted or unchanged start_idx and end_idx
 */
__device__ __inline__ void trim_field_start_end(const char *data,
                                                long *start,
                                                long *end,
                                                char quotechar = '\0')
{
  while ((*start < *end) && is_whitespace(data[*start])) { (*start)++; }
  if ((*start < *end) && data[*start] == quotechar) { (*start)++; }
  while ((*start <= *end) && is_whitespace(data[*end])) { (*end)--; }
  if ((*start <= *end) && data[*end] == quotechar) { (*end)--; }
}

/*
 * @brief Returns true is the input character is a valid digit.
 * Supports both decimal and hexadecimal digits (uppercase and lowercase).
 *
 * @param c Character to check
 * @param is_hex Whether to check as a hexadecimal
 *
 * @return `true` if it is digit-like, `false` otherwise
 */
__device__ __inline__ bool is_digit(char c, bool is_hex = false)
{
  if (c >= '0' && c <= '9') return true;

  if (is_hex) {
    if (c >= 'A' && c <= 'F') return true;
    if (c >= 'a' && c <= 'f') return true;
  }

  return false;
}

/*
 * @brief Checks whether the given character counters indicate a potentially
 * valid date and/or time field.
 *
 * For performance and simplicity, we detect only the most common date
 * formats. Example formats that are detectable:
 *
 *    `2001/02/30`
 *    `2001-02-30 00:00:00`
 *    `2/30/2001 T04:05:60.7`
 *    `2 / 1 / 2011`
 *    `02/January`
 *
 * @param len Number of non special-symbol or numeric characters
 * @param decimal_count Number of '.' characters
 * @param colon_count Number of ':' characters
 * @param dash_count Number of '-' characters
 * @param slash_count Number of '/' characters
 *
 * @return `true` if it is date-like, `false` otherwise
 */
__device__ __inline__ bool is_datetime(
  long len, long decimal_count, long colon_count, long dash_count, long slash_count)
{
  // Must not exceed count of longest month (September) plus `T` time indicator
  if (len > 10) { return false; }
  // Must not exceed more than one decimals or more than two time separators
  if (decimal_count > 1 || colon_count > 2) { return false; }
  // Must have one or two '-' or '/' but not both as date separators
  if ((dash_count > 0 && dash_count < 3 && slash_count == 0) ||
      (dash_count == 0 && slash_count > 0 && slash_count < 3)) {
    return true;
  }

  return false;
}

/*
 * @brief Returns true if the counters indicate a potentially valid float.
 * False positives are possible because positions are not taken into account.
 * For example, field "e.123-" would match the pattern.
 *
 * @param len Number of non special-symbol or numeric characters
 * @param digit_count Number of digits characters
 * @param decimal_count Number of '.' characters
 * @param dash_count Number of '-' characters
 * @param exponent_count Number of 'e or E' characters
 *
 * @return `true` if it is floating point-like, `false` otherwise
 */
__device__ __inline__ bool is_floatingpoint(
  long len, long digit_count, long decimal_count, long dash_count, long exponent_count)
{
  // Can't have more than one exponent and one decimal point
  if (decimal_count > 1) return false;
  if (exponent_count > 1) return false;

  // Without the exponent or a decimal point, this is an integer, not a float
  if (decimal_count == 0 && exponent_count == 0) return false;

  // Can only have one '-' per component
  if (dash_count > 1 + exponent_count) return false;

  // If anything other than these characters is present, it's not a float
  if (digit_count + decimal_count + dash_count + exponent_count != len) { return false; }

  // Needs at least 1 digit, 2 if exponent is present
  if (digit_count < 1 + exponent_count) return false;

  return true;
}

/*
 * @brief CUDA kernel that parses and converts CSV data into cuDF column data.
 *
 * Data is processed in one row/record at a time, so the number of total
 * threads (tid) is equal to the number of rows.
 *
 * @param opts A set of parsing options
 * @param csv_text The entire CSV data to read
 * @param column_flags Per-column parsing behavior flags
 * @param row_offsets The start the CSV data of interest
 * @param d_columnData The count for each column data type
 */
__global__ void __launch_bounds__(csvparse_block_dim)
  data_type_detection(parse_options_view const opts,
                      device_span<char const> csv_text,
                      device_span<column_parse::flags const> const column_flags,
                      device_span<uint64_t const> const row_offsets,
                      device_span<column_parse::stats> d_columnData)
{
  auto raw_csv = csv_text.data();

  // ThreadIds range per block, so also need the blockId
  // This is entry into the fields; threadId is an element within `num_records`
  long rec_id      = threadIdx.x + (blockDim.x * blockIdx.x);
  long rec_id_next = rec_id + 1;

  // we can have more threads than data, make sure we are not past the end of
  // the data
  if (rec_id_next >= row_offsets.size()) { return; }

  long start = row_offsets[rec_id];
  long stop  = row_offsets[rec_id_next];

  long pos       = start;
  int col        = 0;
  int actual_col = 0;

  // Going through all the columns of a given record
  while (col < column_flags.size()) {
    if (start > stop) { break; }

    pos = cudf::io::gpu::seek_field_end(raw_csv + pos, raw_csv + stop, opts) - raw_csv;

    // Checking if this is a column that the user wants --- user can filter
    // columns
    if (column_flags[col] & column_parse::enabled) {
      long tempPos   = pos - 1;
      long field_len = pos - start;

      if (field_len <= 0 || serialized_trie_contains(opts.trie_na, raw_csv + start, field_len)) {
        atomicAdd(&d_columnData[actual_col].countNULL, 1);
      } else if (serialized_trie_contains(opts.trie_true, raw_csv + start, field_len) ||
                 serialized_trie_contains(opts.trie_false, raw_csv + start, field_len)) {
        atomicAdd(&d_columnData[actual_col].countBool, 1);
      } else if (cudf::io::gpu::is_infinity(raw_csv + start, raw_csv + tempPos)) {
        atomicAdd(&d_columnData[actual_col].countFloat, 1);
      } else {
        long countNumber   = 0;
        long countDecimal  = 0;
        long countSlash    = 0;
        long countDash     = 0;
        long countPlus     = 0;
        long countColon    = 0;
        long countString   = 0;
        long countExponent = 0;

        // Modify start & end to ignore whitespace and quotechars
        // This could possibly result in additional empty fields
        trim_field_start_end(raw_csv, &start, &tempPos);
        field_len = tempPos - start + 1;

        for (long startPos = start; startPos <= tempPos; startPos++) {
          if (is_digit(raw_csv[startPos])) {
            countNumber++;
            continue;
          }
          // Looking for unique characters that will help identify column types.
          switch (raw_csv[startPos]) {
            case '.': countDecimal++; break;
            case '-': countDash++; break;
            case '+': countPlus++; break;
            case '/': countSlash++; break;
            case ':': countColon++; break;
            case 'e':
            case 'E':
              if (startPos > start && startPos < tempPos) countExponent++;
              break;
            default: countString++; break;
          }
        }

        // Integers have to have the length of the string
        long int_req_number_cnt = field_len;
        // Off by one if they start with a minus sign
        if ((raw_csv[start] == '-' || raw_csv[start] == '+') && field_len > 1) {
          --int_req_number_cnt;
        }

        if (field_len == 0) {
          // Ignoring whitespace and quotes can result in empty fields
          atomicAdd(&d_columnData[actual_col].countNULL, 1);
        } else if (column_flags[col] & column_parse::as_datetime) {
          // PANDAS uses `object` dtype if the date is unparseable
          if (is_datetime(countString, countDecimal, countColon, countDash, countSlash)) {
            atomicAdd(&d_columnData[actual_col].countDateAndTime, 1);
          } else {
            atomicAdd(&d_columnData[actual_col].countString, 1);
          }
        } else if (countNumber == int_req_number_cnt) {
          atomicAdd(&d_columnData[actual_col].countInt64, 1);
        } else if (is_floatingpoint(
                     field_len, countNumber, countDecimal, countDash + countPlus, countExponent)) {
          atomicAdd(&d_columnData[actual_col].countFloat, 1);
        } else {
          atomicAdd(&d_columnData[actual_col].countString, 1);
        }
      }
      actual_col++;
    }
    pos++;
    start = pos;
    col++;
  }
}

template <typename T, int base>
__inline__ __device__ T decode_value(char const *begin,
                                     char const *end,
                                     parse_options_view const &opts)
{
  return cudf::io::gpu::parse_numeric<T, base>(begin, end, opts);
}

template <typename T>
__inline__ __device__ T decode_value(char const *begin,
                                     char const *end,
                                     parse_options_view const &opts)
{
  return cudf::io::gpu::parse_numeric<T>(begin, end, opts);
}

template <>
__inline__ __device__ cudf::timestamp_D decode_value(char const *begin,
                                                     char const *end,
                                                     parse_options_view const &opts)
{
  return timestamp_D{cudf::duration_D{parseDateFormat(begin, end, opts.dayfirst)}};
}

template <>
__inline__ __device__ cudf::timestamp_s decode_value(char const *begin,
                                                     char const *end,
                                                     parse_options_view const &opts)
{
  auto milli = parseDateTimeFormat(begin, end, opts.dayfirst);
  return timestamp_s{cudf::duration_s{milli / 1000}};
}

template <>
__inline__ __device__ cudf::timestamp_ms decode_value(char const *begin,
                                                      char const *end,
                                                      parse_options_view const &opts)
{
  auto milli = parseDateTimeFormat(begin, end, opts.dayfirst);
  return timestamp_ms{cudf::duration_ms{milli}};
}

template <>
__inline__ __device__ cudf::timestamp_us decode_value(char const *begin,
                                                      char const *end,
                                                      parse_options_view const &opts)
{
  auto milli = parseDateTimeFormat(begin, end, opts.dayfirst);
  return timestamp_us{cudf::duration_us{milli * 1000}};
}

template <>
__inline__ __device__ cudf::timestamp_ns decode_value(char const *begin,
                                                      char const *end,
                                                      parse_options_view const &opts)
{
  auto milli = parseDateTimeFormat(begin, end, opts.dayfirst);
  return timestamp_ns{cudf::duration_ns{milli * 1000000}};
}

#ifndef DURATION_DECODE_VALUE
#define DURATION_DECODE_VALUE(Type)                                     \
  template <>                                                           \
  __inline__ __device__ Type decode_value(                              \
    const char *begin, const char *end, parse_options_view const &opts) \
  {                                                                     \
    return Type{parseTimeDeltaFormat<Type>(begin, 0, end - begin)};     \
  }
#endif
DURATION_DECODE_VALUE(duration_D)
DURATION_DECODE_VALUE(duration_s)
DURATION_DECODE_VALUE(duration_ms)
DURATION_DECODE_VALUE(duration_us)
DURATION_DECODE_VALUE(duration_ns)

// The purpose of this is merely to allow compilation ONLY
// TODO : make this work for csv
template <>
__inline__ __device__ cudf::string_view decode_value(char const *begin,
                                                     char const *end,
                                                     parse_options_view const &opts)
{
  return cudf::string_view{};
}

// The purpose of this is merely to allow compilation ONLY
template <>
__inline__ __device__ cudf::dictionary32 decode_value(char const *begin,
                                                      char const *end,
                                                      parse_options_view const &opts)
{
  return cudf::dictionary32{};
}

// The purpose of this is merely to allow compilation ONLY
// TODO : make this work for csv
template <>
__inline__ __device__ cudf::list_view decode_value(char const *begin,
                                                   char const *end,
                                                   parse_options_view const &opts)
{
  return cudf::list_view{};
}

// The purpose of this is merely to allow compilation ONLY
// TODO : make this work for csv
template <>
__inline__ __device__ numeric::decimal32 decode_value(char const *begin,
                                                      char const *end,
                                                      parse_options_view const &opts)
{
  return numeric::decimal32{};
}

// The purpose of this is merely to allow compilation ONLY
// TODO : make this work for csv
template <>
__inline__ __device__ numeric::decimal64 decode_value(char const *begin,
                                                      char const *end,
                                                      parse_options_view const &opts)
{
  return numeric::decimal64{};
}

// The purpose of this is merely to allow compilation ONLY
// TODO : make this work for csv
template <>
__inline__ __device__ cudf::struct_view decode_value(char const *begin,
                                                     char const *end,
                                                     parse_options_view const &opts)
{
  return cudf::struct_view{};
}

/**
 * @brief Functor for converting CSV raw data to typed value.
 */
struct decode_op {
  /**
   * @brief Dispatch for numeric types whose values can be convertible to
   * 0 or 1 to represent boolean false/true, based upon checking against a
   * true/false values list.
   *
   * @return bool Whether the parsed value is valid.
   */
  template <typename T,
            typename std::enable_if_t<std::is_integral<T>::value and !std::is_same<T, bool>::value>
              * = nullptr>
  __host__ __device__ __forceinline__ bool operator()(void *out_buffer,
                                                      size_t row,
                                                      char const *begin,
                                                      char const *end,
                                                      parse_options_view const &opts,
                                                      column_parse::flags flags)
  {
    static_cast<T *>(out_buffer)[row] = [&]() {
      // Check for user-specified true/false values first, where the output is
      // replaced with 1/0 respectively
      const size_t field_len = end - begin + 1;
      if (serialized_trie_contains(opts.trie_true, begin, field_len)) {
        return static_cast<T>(1);
      } else if (serialized_trie_contains(opts.trie_false, begin, field_len)) {
        return static_cast<T>(0);
      } else {
        if (flags & column_parse::as_hexadecimal) {
          return decode_value<T, 16>(begin, end, opts);
        } else {
          return decode_value<T>(begin, end, opts);
        }
      }
    }();

    return true;
  }

  /**
   * @brief Dispatch for boolean type types.
   */
  template <typename T, typename std::enable_if_t<std::is_same<T, bool>::value> * = nullptr>
  __host__ __device__ __forceinline__ bool operator()(void *out_buffer,
                                                      size_t row,
                                                      char const *begin,
                                                      char const *end,
                                                      parse_options_view const &opts,
                                                      column_parse::flags flags)
  {
    auto &value{static_cast<T *>(out_buffer)[row]};

    // Check for user-specified true/false values first, where the output is
    // replaced with 1/0 respectively
    const size_t field_len = end - begin + 1;
    if (serialized_trie_contains(opts.trie_true, begin, field_len)) {
      value = 1;
    } else if (serialized_trie_contains(opts.trie_false, begin, field_len)) {
      value = 0;
    } else {
      value = decode_value<T>(begin, end, opts);
    }
    return true;
  }

  /**
   * @brief Dispatch for floating points, which are set to NaN if the input
   * is not valid. In such case, the validity mask is set to zero too.
   */
  template <typename T, typename std::enable_if_t<std::is_floating_point<T>::value> * = nullptr>
  __host__ __device__ __forceinline__ bool operator()(void *out_buffer,
                                                      size_t row,
                                                      char const *begin,
                                                      char const *end,
                                                      parse_options_view const &opts,
                                                      column_parse::flags flags)
  {
    auto &value{static_cast<T *>(out_buffer)[row]};

    value = decode_value<T>(begin, end, opts);
    return !std::isnan(value);
  }

  /**
   * @brief Dispatch for all other types.
   */
  template <typename T,
            typename std::enable_if_t<!std::is_integral<T>::value and
                                      !std::is_floating_point<T>::value> * = nullptr>
  __host__ __device__ __forceinline__ bool operator()(void *out_buffer,
                                                      size_t row,
                                                      char const *begin,
                                                      char const *end,
                                                      parse_options_view const &opts,
                                                      column_parse::flags flags)
  {
    auto &value{static_cast<T *>(out_buffer)[row]};

    value = decode_value<T>(begin, end, opts);
    return true;
  }
};

/**
 * @brief CUDA kernel that parses and converts CSV data into cuDF column data.
 *
 * Data is processed one record at a time
 *
 * @param[in] raw_csv The entire CSV data to read
 * @param[in] opts A set of parsing options
 * @param[in] num_records The number of lines/rows of CSV data
 * @param[in] num_columns The number of columns of CSV data
 * @param[in] column_flags Per-column parsing behavior flags
 * @param[in] recStart The start the CSV data of interest
 * @param[in] dtype The data type of the column
 * @param[out] data The output column data
 * @param[out] valid The bitmaps indicating whether column fields are valid
 * @param[out] num_valid The numbers of valid fields in columns
 **/
__global__ void __launch_bounds__(csvparse_block_dim)
  convert_csv_to_cudf(cudf::io::parse_options_view options,
                      device_span<char const> data,
                      device_span<column_parse::flags const> column_flags,
                      device_span<uint64_t const> row_offsets,
                      device_span<cudf::data_type const> dtypes,
                      device_span<void *> columns,
                      device_span<cudf::bitmask_type *> valids)
{
  auto raw_csv = data.data();
  // thread IDs range per block, so also need the block id.
  // this is entry into the field array - tid is an elements within the num_entries array
  long rec_id      = threadIdx.x + (blockDim.x * blockIdx.x);
  long rec_id_next = rec_id + 1;

  // we can have more threads than data, make sure we are not past the end of
  // the data
  if (rec_id_next >= row_offsets.size()) return;

  long start = row_offsets[rec_id];
  long stop  = row_offsets[rec_id_next];

  long pos       = start;
  int col        = 0;
  int actual_col = 0;

  while (col < column_flags.size()) {
    if (start > stop) break;

    pos = cudf::io::gpu::seek_field_end(raw_csv + pos, raw_csv + stop, options) - raw_csv;

    if (column_flags[col] & column_parse::enabled) {
      // check if the entire field is a NaN string - consistent with pandas
      const bool is_na = serialized_trie_contains(options.trie_na, raw_csv + start, pos - start);

      // Modify start & end to ignore whitespace and quotechars
      long tempPos = pos - 1;
      if (!is_na && dtypes[actual_col].id() != cudf::type_id::STRING) {
        trim_field_start_end(raw_csv, &start, &tempPos, options.quotechar);
      }

      if (!is_na && start <= (tempPos)) {  // Empty fields are not legal values

        // Type dispatcher does not handle STRING
        if (dtypes[actual_col].id() == cudf::type_id::STRING) {
          long end = pos;
          if (options.keepquotes == false) {
            if ((raw_csv[start] == options.quotechar) && (raw_csv[end - 1] == options.quotechar)) {
              start++;
              end--;
            }
          }
          auto str_list = static_cast<std::pair<const char *, size_t> *>(columns[actual_col]);
          str_list[rec_id].first  = raw_csv + start;
          str_list[rec_id].second = end - start;
        } else {
          if (cudf::type_dispatcher(dtypes[actual_col],
                                    decode_op{},
                                    columns[actual_col],
                                    rec_id,
                                    raw_csv + start,
                                    raw_csv + tempPos,
                                    options,
                                    column_flags[col])) {
            // set the valid bitmap - all bits were set to 0 to start
            set_bit(valids[actual_col], rec_id);
          }
        }
      } else if (dtypes[actual_col].id() == cudf::type_id::STRING) {
        auto str_list = static_cast<std::pair<const char *, size_t> *>(columns[actual_col]);
        str_list[rec_id].first  = nullptr;
        str_list[rec_id].second = 0;
      }
      actual_col++;
    }
    pos++;
    start = pos;
    col++;
  }
}

/*
 * @brief Merge two packed row contexts (each corresponding to a block of characters)
 * and return the packed row context corresponding to the merged character block
 **/
inline __device__ packed_rowctx_t merge_row_contexts(packed_rowctx_t first_ctx,
                                                     packed_rowctx_t second_ctx)
{
  uint32_t id0 = get_row_context(first_ctx, ROW_CTX_NONE) & 3;
  uint32_t id1 = get_row_context(first_ctx, ROW_CTX_QUOTE) & 3;
  uint32_t id2 = get_row_context(first_ctx, ROW_CTX_COMMENT) & 3;
  return (first_ctx & ~pack_row_contexts(3, 3, 3)) +
         pack_row_contexts(get_row_context(second_ctx, id0),
                           get_row_context(second_ctx, id1),
                           get_row_context(second_ctx, id2));
}

/*
 * @brief Per-character context:
 * 1-bit count (0 or 1) per context in the lower 4 bits
 * 2-bit output context id per input context in bits 8..15
 **/
constexpr __device__ uint32_t make_char_context(uint32_t id0,
                                                uint32_t id1,
                                                uint32_t id2 = ROW_CTX_COMMENT,
                                                uint32_t c0  = 0,
                                                uint32_t c1  = 0,
                                                uint32_t c2  = 0)
{
  return (id0 << 8) | (id1 << 10) | (id2 << 12) | (ROW_CTX_EOF << 14) | (c0) | (c1 << 1) |
         (c2 << 2);
}

/*
 * @brief Merge a 1-character context to keep track of bitmasks where new rows occur
 * Merges a single-character "block" row context at position pos with the current
 * block's row context (the current block contains 32-pos characters)
 *
 * @param ctx Current block context and new rows bitmaps
 * @param char_ctx state transitions associated with new character
 * @param pos Position within the current 32-character block
 *
 * NOTE: This is probably the most performance-critical piece of the row gathering kernel.
 * The char_ctx value should be created via make_char_context, and its value should
 * have been evaluated at compile-time.
 *
 **/
inline __device__ void merge_char_context(uint4 &ctx, uint32_t char_ctx, uint32_t pos)
{
  uint32_t id0 = (ctx.w >> 0) & 3;
  uint32_t id1 = (ctx.w >> 2) & 3;
  uint32_t id2 = (ctx.w >> 4) & 3;
  // Set the newrow bit in the bitmap at the corresponding position
  ctx.x |= ((char_ctx >> id0) & 1) << pos;
  ctx.y |= ((char_ctx >> id1) & 1) << pos;
  ctx.z |= ((char_ctx >> id2) & 1) << pos;
  // Update the output context ids
  ctx.w = ((char_ctx >> (8 + id0 * 2)) & 0x03) | ((char_ctx >> (6 + id1 * 2)) & 0x0c) |
          ((char_ctx >> (4 + id2 * 2)) & 0x30) | (ROW_CTX_EOF << 6);
}

/*
 * Convert the context-with-row-bitmaps version to a packed row context
 **/
inline __device__ packed_rowctx_t pack_rowmaps(uint4 ctx_map)
{
  return pack_row_contexts(make_row_context(__popc(ctx_map.x), (ctx_map.w >> 0) & 3),
                           make_row_context(__popc(ctx_map.y), (ctx_map.w >> 2) & 3),
                           make_row_context(__popc(ctx_map.z), (ctx_map.w >> 4) & 3));
}

/*
 * Selects the row bitmap corresponding to the given parser state
 **/
inline __device__ uint32_t select_rowmap(uint4 ctx_map, uint32_t ctxid)
{
  return (ctxid == ROW_CTX_NONE)
           ? ctx_map.x
           : (ctxid == ROW_CTX_QUOTE) ? ctx_map.y : (ctxid == ROW_CTX_COMMENT) ? ctx_map.z : 0;
}

/**
 * @brief Single pair-wise 512-wide row context merge transform
 *
 * Merge row context blocks and record the merge operation in a context
 * tree so that the transform is reversible.
 * The tree is organized such that the left and right children of node n
 * are located at indices n*2 and n*2+1, the root node starting at index 1
 *
 * @tparam lanemask mask to specify source of packed row context
 * @tparam tmask mask to specify principle thread for merging row context
 * @tparam base start location for writing into packed row context tree
 * @tparam level_scale level of the node in the tree
 * @param ctxtree[out] packed row context tree
 * @param ctxb[in] packed row context for the current character block
 * @param t thread id (leaf node id)
 *
 */
template <uint32_t lanemask, uint32_t tmask, uint32_t base, uint32_t level_scale>
inline __device__ void ctx_merge(uint64_t *ctxtree, packed_rowctx_t *ctxb, uint32_t t)
{
  uint64_t tmp = SHFL_XOR(*ctxb, lanemask);
  if (!(t & tmask)) {
    *ctxb                              = merge_row_contexts(*ctxb, tmp);
    ctxtree[base + (t >> level_scale)] = *ctxb;
  }
}

/**
 * @brief Single 512-wide row context inverse merge transform
 *
 * Walks the context tree starting from a root node
 *
 * @tparam rmask Mask to specify which threads write input row context
 * @param[in] base Start read location of the merge transform tree
 * @param[in] ctxtree Merge transform tree
 * @param[in] ctx Input context
 * @param[in] brow4 output row in block *4
 * @param[in] t thread id (leaf node id)
 */
template <uint32_t rmask>
inline __device__ void ctx_unmerge(
  uint32_t base, uint64_t *ctxtree, uint32_t *ctx, uint32_t *brow4, uint32_t t)
{
  rowctx32_t ctxb_left, ctxb_right, ctxb_sum;
  ctxb_sum   = get_row_context(ctxtree[base], *ctx);
  ctxb_left  = get_row_context(ctxtree[(base)*2 + 0], *ctx);
  ctxb_right = get_row_context(ctxtree[(base)*2 + 1], ctxb_left & 3);
  if (t & (rmask)) {
    *brow4 += (ctxb_sum & ~3) - (ctxb_right & ~3);
    *ctx = ctxb_left & 3;
  }
}

/*
 * @brief 512-wide row context merge transform
 *
 * Repeatedly merge row context blocks, keeping track of each merge operation
 * in a context tree so that the transform is reversible
 * The tree is organized such that the left and right children of node n
 * are located at indices n*2 and n*2+1, the root node starting at index 1
 *
 * Each node contains the counts and output contexts corresponding to the
 * possible input contexts.
 * Each parent node's count is obtained by adding the corresponding counts
 * from the left child node with the right child node's count selected from
 * the left child node's output context:
 *   parent.count[k] = left.count[k] + right.count[left.outctx[k]]
 *   parent.outctx[k] = right.outctx[left.outctx[k]]
 *
 * @param ctxtree[out] packed row context tree
 * @param ctxb[in] packed row context for the current character block
 * @param t thread id (leaf node id)
 *
 **/
static inline __device__ void rowctx_merge_transform(uint64_t ctxtree[1024],
                                                     packed_rowctx_t ctxb,
                                                     uint32_t t)
{
  ctxtree[512 + t] = ctxb;
  ctx_merge<1, 0x1, 256, 1>(ctxtree, &ctxb, t);
  ctx_merge<2, 0x3, 128, 2>(ctxtree, &ctxb, t);
  ctx_merge<4, 0x7, 64, 3>(ctxtree, &ctxb, t);
  ctx_merge<8, 0xf, 32, 4>(ctxtree, &ctxb, t);
  __syncthreads();
  if (t < 32) {
    ctxb = ctxtree[32 + t];
    ctx_merge<1, 0x1, 16, 1>(ctxtree, &ctxb, t);
    ctx_merge<2, 0x3, 8, 2>(ctxtree, &ctxb, t);
    ctx_merge<4, 0x7, 4, 3>(ctxtree, &ctxb, t);
    ctx_merge<8, 0xf, 2, 4>(ctxtree, &ctxb, t);
    // Final stage
    uint64_t tmp = SHFL_XOR(ctxb, 16);
    if (t == 0) { ctxtree[1] = merge_row_contexts(ctxb, tmp); }
  }
}

/*
 * @brief 512-wide row context inverse merge transform
 *
 * Walks the context tree starting from the root node (index 1) using
 * the starting context in node index 0.
 * The return value is the starting row and input context for the given leaf node
 *
 * @param[in] ctxtree Merge transform tree
 * @param[in] t thread id (leaf node id)
 *
 * @return Final row context and count (row_position*4 + context_id format)
 **/
static inline __device__ rowctx32_t rowctx_inverse_merge_transform(uint64_t ctxtree[1024],
                                                                   uint32_t t)
{
  uint32_t ctx     = ctxtree[0] & 3;  // Starting input context
  rowctx32_t brow4 = 0;               // output row in block *4

  ctx_unmerge<256>(1, ctxtree, &ctx, &brow4, t);
  ctx_unmerge<128>(2 + (t >> 8), ctxtree, &ctx, &brow4, t);
  ctx_unmerge<64>(4 + (t >> 7), ctxtree, &ctx, &brow4, t);
  ctx_unmerge<32>(8 + (t >> 6), ctxtree, &ctx, &brow4, t);
  ctx_unmerge<16>(16 + (t >> 5), ctxtree, &ctx, &brow4, t);
  ctx_unmerge<8>(32 + (t >> 4), ctxtree, &ctx, &brow4, t);
  ctx_unmerge<4>(64 + (t >> 3), ctxtree, &ctx, &brow4, t);
  ctx_unmerge<2>(128 + (t >> 2), ctxtree, &ctx, &brow4, t);
  ctx_unmerge<1>(256 + (t >> 1), ctxtree, &ctx, &brow4, t);

  return brow4 + ctx;
}

/**
 * @brief Gather row offsets from CSV character data split into 16KB chunks
 *
 * This is done in two phases: the first phase returns the possible row counts
 * per 16K character block for each possible parsing context at the start of the block,
 * along with the resulting parsing context at the end of the block.
 * The caller can then compute the actual parsing context at the beginning of each
 * individual block and total row count.
 * The second phase outputs the location of each row in the block, using the parsing
 * context and initial row counter accumulated from the results of the previous phase.
 * Row parsing context will be updated after phase 2 such that the value contains
 * the number of rows starting at byte_range_end or beyond.
 *
 * @param row_ctx Row parsing context (output of phase 1 or input to phase 2)
 * @param offsets_out Row offsets (nullptr for phase1, non-null indicates phase 2)
 * @param data Base pointer of character data (all row offsets are relative to this)
 * @param chunk_size Total number of characters to parse
 * @param parse_pos Current parsing position in the file
 * @param start_offset Position of the start of the character buffer in the file
 * @param data_size CSV file size
 * @param byte_range_start Ignore rows starting before this position in the file
 * @param byte_range_end In phase 2, store the number of rows beyond range in row_ctx
 * @param skip_rows Number of rows to skip (ignored in phase 1)
 * @param terminator Line terminator character
 * @param delimiter Column delimiter character
 * @param quotechar Quote character
 * @param escapechar Delimiter escape character
 * @param commentchar Comment line character (skip rows starting with this character)
 **/
__global__ void __launch_bounds__(rowofs_block_dim)
  gather_row_offsets_gpu(uint64_t *row_ctx,
                         device_span<uint64_t> offsets_out,
                         device_span<char const> const data,
                         size_t chunk_size,
                         size_t parse_pos,
                         size_t start_offset,
                         size_t data_size,
                         size_t byte_range_start,
                         size_t byte_range_end,
                         size_t skip_rows,
                         int terminator,
                         int delimiter,
                         int quotechar,
                         int escapechar,
                         int commentchar)
{
  auto start = data.begin();
  __shared__ __align__(8) uint64_t ctxtree[rowofs_block_dim * 2];
  using warp_reduce      = typename hipcub::WarpReduce<uint32_t>;
  using half_warp_reduce = typename hipcub::WarpReduce<uint32_t, 16>;
  __shared__ union {
    typename warp_reduce::TempStorage full;
    typename half_warp_reduce::TempStorage half[rowofs_block_dim / 32];
  } temp_storage;

  const char *end = start + (min(parse_pos + chunk_size, data_size) - start_offset);
  uint32_t t      = threadIdx.x;
  size_t block_pos =
    (parse_pos - start_offset) + blockIdx.x * static_cast<size_t>(rowofs_block_bytes) + t * 32;
  const char *cur = start + block_pos;

  // Initial state is neutral context (no state transitions), zero rows
  uint4 ctx_map = {
    .x = 0,
    .y = 0,
    .z = 0,
    .w = (ROW_CTX_NONE << 0) | (ROW_CTX_QUOTE << 2) | (ROW_CTX_COMMENT << 4) | (ROW_CTX_EOF << 6)};
  int c, c_prev = (cur > start && cur <= end) ? cur[-1] : terminator;
  // Loop through all 32 bytes and keep a bitmask of row starts for each possible input context
  for (uint32_t pos = 0; pos < 32; pos++, cur++, c_prev = c) {
    uint32_t ctx;
    if (cur < end) {
      c = cur[0];
      if (c_prev == terminator) {
        if (c == commentchar) {
          // Start of a new comment row
          ctx = make_char_context(ROW_CTX_COMMENT, ROW_CTX_QUOTE, ROW_CTX_COMMENT, 1, 0, 1);
        } else if (c == quotechar) {
          // Quoted string on newrow, or quoted string ending in terminator
          ctx = make_char_context(ROW_CTX_QUOTE, ROW_CTX_NONE, ROW_CTX_QUOTE, 1, 0, 1);
        } else {
          // Start of a new row unless within a quote
          ctx = make_char_context(ROW_CTX_NONE, ROW_CTX_QUOTE, ROW_CTX_NONE, 1, 0, 1);
        }
      } else if (c == quotechar) {
        if (c_prev == delimiter || c_prev == quotechar) {
          // Quoted string after delimiter, quoted string ending in delimiter, or double-quote
          ctx = make_char_context(ROW_CTX_QUOTE, ROW_CTX_NONE);
        } else {
          // Closing or ignored quote
          ctx = make_char_context(ROW_CTX_NONE, ROW_CTX_NONE);
        }
      } else {
        // Neutral character
        ctx = make_char_context(ROW_CTX_NONE, ROW_CTX_QUOTE);
      }
    } else {
      const char *data_end = start + data_size - start_offset;
      if (cur <= end && cur == data_end) {
        // Add a newline at data end (need the extra row offset to infer length of previous row)
        ctx = make_char_context(ROW_CTX_EOF, ROW_CTX_EOF, ROW_CTX_EOF, 1, 1, 1);
      } else {
        // Pass-through context (beyond chunk_size or data_end)
        ctx = make_char_context(ROW_CTX_NONE, ROW_CTX_QUOTE, ROW_CTX_COMMENT);
      }
    }
    // Merge with current context, keeping track of where new rows occur
    merge_char_context(ctx_map, ctx, pos);
  }

  // Eliminate rows that start before byte_range_start
  if (start_offset + block_pos < byte_range_start) {
    uint32_t dist_minus1 = min(byte_range_start - (start_offset + block_pos) - 1, UINT64_C(31));
    uint32_t mask        = 0xfffffffe << dist_minus1;
    ctx_map.x &= mask;
    ctx_map.y &= mask;
    ctx_map.z &= mask;
  }

  // Convert the long-form {rowmap,outctx}[inctx] version into packed version
  // {rowcount,ouctx}[inctx], then merge the row contexts of the 32-character blocks into
  // a single 16K-character block context
  rowctx_merge_transform(ctxtree, pack_rowmaps(ctx_map), t);

  // If this is the second phase, get the block's initial parser state and row counter
  if (offsets_out.data()) {
    if (t == 0) { ctxtree[0] = row_ctx[blockIdx.x]; }
    __syncthreads();

    // Walk back the transform tree with the known initial parser state
    rowctx32_t ctx             = rowctx_inverse_merge_transform(ctxtree, t);
    uint64_t row               = (ctxtree[0] >> 2) + (ctx >> 2);
    uint32_t rows_out_of_range = 0;
    uint32_t rowmap            = select_rowmap(ctx_map, ctx & 3);
    // Output row positions
    while (rowmap != 0) {
      uint32_t pos = __ffs(rowmap);
      block_pos += pos;
      if (row >= skip_rows && row - skip_rows < offsets_out.size()) {
        // Output byte offsets are relative to the base of the input buffer
        offsets_out[row - skip_rows] = block_pos - 1;
        rows_out_of_range += (start_offset + block_pos - 1 >= byte_range_end);
      }
      row++;
      rowmap >>= pos;
    }
    // Return the number of rows out of range
    rows_out_of_range = half_warp_reduce(temp_storage.half[t / 32]).Sum(rows_out_of_range);
    __syncthreads();
    if (!(t & 0xf)) { ctxtree[t >> 4] = rows_out_of_range; }
    __syncthreads();
    if (t < 32) {
      rows_out_of_range = warp_reduce(temp_storage.full).Sum(static_cast<uint32_t>(ctxtree[t]));
      if (t == 0) { row_ctx[blockIdx.x] = rows_out_of_range; }
    }
  } else {
    // Just store the row counts and output contexts
    if (t == 0) { row_ctx[blockIdx.x] = ctxtree[1]; }
  }
}

size_t __host__ count_blank_rows(const cudf::io::parse_options_view &opts,
                                 device_span<char const> const data,
                                 device_span<uint64_t const> const row_offsets,
                                 hipStream_t stream)
{
  const auto newline  = opts.skipblanklines ? opts.terminator : opts.comment;
  const auto comment  = opts.comment != '\0' ? opts.comment : newline;
  const auto carriage = (opts.skipblanklines && opts.terminator == '\n') ? '\r' : comment;
  return thrust::count_if(
    rmm::exec_policy(stream)->on(stream),
    row_offsets.begin(),
    row_offsets.end(),
    [data = data, newline, comment, carriage] __device__(const uint64_t pos) {
      return ((pos != data.size()) &&
              (data[pos] == newline || data[pos] == comment || data[pos] == carriage));
    });
}

void __host__ remove_blank_rows(cudf::io::parse_options_view const &options,
                                device_span<char const> const data,
                                rmm::device_vector<uint64_t> &row_offsets,
                                hipStream_t stream)
{
  size_t d_size       = data.size();
  const auto newline  = options.skipblanklines ? options.terminator : options.comment;
  const auto comment  = options.comment != '\0' ? options.comment : newline;
  const auto carriage = (options.skipblanklines && options.terminator == '\n') ? '\r' : comment;
  auto new_end        = thrust::remove_if(
    rmm::exec_policy(stream)->on(stream),
    row_offsets.begin(),
    row_offsets.end(),
    [data = data, d_size, newline, comment, carriage] __device__(const uint64_t pos) {
      return ((pos != d_size) &&
              (data[pos] == newline || data[pos] == comment || data[pos] == carriage));
    });
  row_offsets.resize(new_end - row_offsets.begin());
}

thrust::host_vector<column_parse::stats> detect_column_types(
  cudf::io::parse_options_view const &options,
  device_span<char const> const data,
  device_span<column_parse::flags const> const column_flags,
  device_span<uint64_t const> const row_starts,
  size_t const num_active_columns,
  hipStream_t stream)
{
  // Calculate actual block count to use based on records count
  const int block_size = csvparse_block_dim;
  const int grid_size  = (row_starts.size() + block_size - 1) / block_size;

  auto d_stats = rmm::device_vector<column_parse::stats>(num_active_columns);

  data_type_detection<<<grid_size, block_size, 0, stream>>>(
    options, data, column_flags, row_starts, d_stats);

  return thrust::host_vector<column_parse::stats>(d_stats);
}

void __host__ decode_row_column_data(cudf::io::parse_options_view const &options,
                                     device_span<char const> const data,
                                     device_span<column_parse::flags const> const column_flags,
                                     device_span<uint64_t const> const row_offsets,
                                     device_span<cudf::data_type const> const dtypes,
                                     device_span<void *> const columns,
                                     device_span<cudf::bitmask_type *> const valids,
                                     hipStream_t stream)
{
  // Calculate actual block count to use based on records count
  auto const block_size = csvparse_block_dim;
  auto const num_rows   = row_offsets.size() - 1;
  auto const grid_size  = (num_rows + block_size - 1) / block_size;

  convert_csv_to_cudf<<<grid_size, block_size, 0, stream>>>(
    options, data, column_flags, row_offsets, dtypes, columns, valids);
}

uint32_t __host__ gather_row_offsets(const parse_options_view &options,
                                     uint64_t *row_ctx,
                                     device_span<uint64_t> const offsets_out,
                                     device_span<char const> const data,
                                     size_t chunk_size,
                                     size_t parse_pos,
                                     size_t start_offset,
                                     size_t data_size,
                                     size_t byte_range_start,
                                     size_t byte_range_end,
                                     size_t skip_rows,
                                     hipStream_t stream)
{
  uint32_t dim_grid = 1 + (chunk_size / rowofs_block_bytes);

  gather_row_offsets_gpu<<<dim_grid, rowofs_block_dim, 0, stream>>>(
    row_ctx,
    offsets_out,
    data,
    chunk_size,
    parse_pos,
    start_offset,
    data_size,
    byte_range_start,
    byte_range_end,
    skip_rows,
    options.terminator,
    options.delimiter,
    (options.quotechar) ? options.quotechar : 0x100,
    /*(options.escapechar) ? options.escapechar :*/ 0x100,
    (options.comment) ? options.comment : 0x100);

  return dim_grid;
}

}  // namespace gpu
}  // namespace csv
}  // namespace io
}  // namespace cudf
