#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2018, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
	 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

/**
 * @file csv-reader.cu  code to read csv data
 *
 * CSV Reader
 */


#include <hip/hip_runtime.h>

#include <algorithm>
#include <iostream>
#include <numeric>
#include <string>
#include <vector>
#include <memory>

#include <stdio.h>
#include <stdlib.h>

#include <unistd.h>
#include <fcntl.h>
#include <sys/types.h>
#include <sys/stat.h>
#include <sys/mman.h>

#include <thrust/scan.h>
#include <thrust/reduce.h>
#include <thrust/device_ptr.h>
#include <thrust/execution_policy.h>

#include <thrust/host_vector.h>

#include "type_conversion.cuh"
#include "datetime_parser.cuh"

#include "cudf.h"
#include "utilities/error_utils.hpp"
#include "utilities/trie.cuh"
#include "utilities/type_dispatcher.hpp"
#include "utilities/cudf_utils.h" 

#include <nvstrings/NVStrings.h>

#include "rmm/rmm.h"
#include "rmm/thrust_rmm_allocator.h"
#include "io/comp/io_uncomp.h"

#include "io/cuio_common.hpp"
#include "io/utilities/parsing_utils.cuh"
#include "io/utilities/wrapper_utils.hpp"

using std::vector;
using std::string;

/**---------------------------------------------------------------------------*
 * @brief Struct used for internal parsing state
 *---------------------------------------------------------------------------**/
typedef struct raw_csv_ {
    device_buffer<char> 	data;		// on-device: the raw unprocessed CSV data - loaded as a large char * array
    device_buffer<uint64_t> recStart;	// on-device: Starting position of the records.

    ParseOptions			opts;			// options to control parsing behavior

    long				num_bytes;		// host: the number of bytes in the data
    long				num_bits;		// host: the number of 64-bit bitmaps (different than valid)
	gdf_size_type 		num_records;	// host: number of records loaded into device memory, and then number of records to read
	// int				num_cols;		// host: number of columns
	int					num_active_cols;// host: number of columns that will be return to user.
	int					num_actual_cols;// host: number of columns in the file --- based on the number of columns in header
    vector<gdf_dtype>	dtypes;			// host: array of dtypes (since gdf_columns are not created until end)
    vector<string>		col_names;		// host: array of column names
	
	thrust::host_vector<bool>	h_parseCol;	// host   : array of booleans stating if column should be parsed in reading process: parseCol[x]=false means that the column x needs to be filtered out.
    rmm::device_vector<bool>	d_parseCol;	// device : array of booleans stating if column should be parsed in reading process: parseCol[x]=false means that the column x needs to be filtered out.

    long        byte_range_offset;  // offset into the data to start parsing
    long        byte_range_size;    // length of the data of interest to parse

    gdf_size_type header_row;       ///< host: Row index of the header
    gdf_size_type nrows;            ///< host: Number of rows to read. -1 for all rows
    gdf_size_type skiprows;         ///< host: Number of rows to skip from the start
    gdf_size_type skipfooter;       ///< host: Number of rows to skip from the end
    std::vector<char> header;       ///< host: Header row data, for parsing column names
    string prefix;                  ///< host: Prepended to column ID if there is no header or input column names

    rmm::device_vector<SerialTrieNode>	d_trueTrie;	// device: serialized trie of values to recognize as true
    rmm::device_vector<SerialTrieNode>	d_falseTrie;// device: serialized trie of values to recognize as false
    rmm::device_vector<SerialTrieNode>	d_naTrie;	// device: serialized trie of NA values
} raw_csv_t;

typedef struct column_data_ {
	unsigned long long countFloat;
	unsigned long long countDateAndTime;
	unsigned long long countString;
	unsigned long long countInt8;
	unsigned long long countInt16;
	unsigned long long countInt32;
	unsigned long long countInt64;
	gdf_size_type countNULL;
} column_data_t;

using string_pair = std::pair<const char*,size_t>;

//
//---------------create and process ---------------------------------------------
//
gdf_error parseArguments(csv_read_arg *args, raw_csv_t *csv);
// gdf_error getColNamesAndTypes(const char **col_names, const  char **dtypes, raw_csv_t *d);
gdf_error inferCompressionType(const char* compression_arg, const char* filepath, string& compression_type);
gdf_error getUncompressedHostData(const char* h_data, size_t num_bytes, 
	const string& compression, 
	vector<char>& h_uncomp_data);
gdf_error uploadDataToDevice(const char* h_uncomp_data, size_t h_uncomp_size, raw_csv_t * raw_csv);

#define checkError(error, txt)  if ( error != GDF_SUCCESS) { std::cerr << "ERROR:  " << error <<  "  in "  << txt << std::endl;  return error; }

//
//---------------CUDA Kernel ---------------------------------------------
//

gdf_error launch_dataConvertColumns(raw_csv_t *raw_csv, void **d_gdf,
                                    gdf_valid_type **valid, gdf_dtype *d_dtypes,
                                    gdf_size_type *num_valid);
gdf_error launch_dataTypeDetection(raw_csv_t *raw_csv,
                                   column_data_t *d_columnData);

__global__ void convertCsvToGdf(char *csv, const ParseOptions opts,
                                gdf_size_type num_records, int num_columns,
                                bool *parseCol, uint64_t *recStart,
                                gdf_dtype *dtype, void **gdf_data,
                                gdf_valid_type **valid,
                                gdf_size_type *num_valid);
__global__ void dataTypeDetection(char *raw_csv, const ParseOptions opts,
                                  gdf_size_type num_records, int num_columns,
                                  bool *parseCol, uint64_t *recStart,
                                  column_data_t *d_columnData);

/**---------------------------------------------------------------------------*
 * @brief Estimates the maximum expected length or a row, based on the number 
 * of columns
 * 
 * If the number of columns is not available, it will return a value large 
 * enough for most use cases
 * 
 * @param[in] num_columns Number of columns in the CSV file (optional)
 * 
 * @return Estimated maximum size of a row, in bytes
 *---------------------------------------------------------------------------**/
 constexpr size_t calculateMaxRowSize(int num_columns=0) noexcept {
	constexpr size_t max_row_bytes = 16*1024; // 16KB
	constexpr size_t column_bytes = 64;
	constexpr size_t base_padding = 1024; // 1KB
	if (num_columns == 0){
		// Use flat size if the number of columns is not known
		return max_row_bytes;
	}
	else {
		// Expand the size based on the number of columns, if available
		return base_padding + num_columns * column_bytes; 
	}
}
/**
* @brief Removes the first and Last quote in the string
*/
string removeQuotes(string str, char quotechar) {
	// Exclude first and last quotation char
	const size_t first_quote = str.find(quotechar);
	if (first_quote != string::npos) {
		str.erase(first_quote, 1);
	}
	const size_t  last_quote = str.rfind(quotechar);
	if (last_quote != string::npos) {
		str.erase(last_quote, 1);
	}

	return str;
}

/**
 * @brief Parse the first row to set the column names in the raw_csv parameter 
 *
 * The first row can be either the header row, or the first data row
 *
 * @param[in,out] raw_csv Structure containing the csv parsing parameters
 * and intermediate results
 * 
 * @return gdf_error with error code on failure, otherwise GDF_SUCCESS
*/
gdf_error setColumnNamesFromCsv(raw_csv_t* raw_csv) {
	vector<char> first_row = raw_csv->header;
	// No header, read the first data row
	if (first_row.empty()) {
		uint64_t first_row_len{};
		// If file only contains one row, raw_csv->recStart[1] is not valid
		if (raw_csv->num_records > 1) {
			CUDA_TRY(hipMemcpy(&first_row_len, raw_csv->recStart.data() + 1, sizeof(uint64_t), hipMemcpyDefault));
		}
		else {
			// File has one row - use the file size for the row size
			first_row_len = raw_csv->num_bytes / sizeof(char);
		}
		first_row.resize(first_row_len);
		CUDA_TRY(hipMemcpy(first_row.data(), raw_csv->data.data(), first_row_len * sizeof(char), hipMemcpyDefault));
	}

	int num_cols = 0;

	bool quotation	= false;
	for (size_t pos = 0, prev = 0; pos < first_row.size(); ++pos) {
		// Flip the quotation flag if current character is a quotechar
		if(first_row[pos] == raw_csv->opts.quotechar) {
			quotation = !quotation;
		}
		// Check if end of a column/row
		else if (pos == first_row.size() - 1 ||
				 (!quotation && first_row[pos] == raw_csv->opts.terminator) ||
				 (!quotation && first_row[pos] == raw_csv->opts.delimiter)) {
			// This is the header, add the column name
			if (raw_csv->header_row >= 0) {
				// Include the current character, in case the line is not terminated
				int col_name_len = pos - prev + 1;
				// Exclude the delimiter/terminator is present
				if (first_row[pos] == raw_csv->opts.delimiter || first_row[pos] == raw_csv->opts.terminator) {
					--col_name_len;
				}
				// Also exclude '\r' character at the end of the column name if it's part of the terminator
				if (col_name_len > 0 &&
					raw_csv->opts.terminator == '\n' &&
					first_row[pos] == '\n' &&
					first_row[pos - 1] == '\r') {
					--col_name_len;
				}

				const string new_col_name(first_row.data() + prev, col_name_len);
				raw_csv->col_names.push_back(removeQuotes(new_col_name, raw_csv->opts.quotechar));

				// Stop parsing when we hit the line terminator; relevant when there is a blank line following the header.
				// In this case, first_row includes multiple line terminators at the end, as the new recStart belongs
				// to a line that comes after the blank line(s)
				if (!quotation && first_row[pos] == raw_csv->opts.terminator){
					break;
				}
			}
			else {
				// This is the first data row, add the automatically generated name
				raw_csv->col_names.push_back(raw_csv->prefix + std::to_string(num_cols));
			}
			num_cols++;

			// Skip adjacent delimiters if delim_whitespace is set
			while (raw_csv->opts.multi_delimiter &&
				   pos < first_row.size() &&
				   first_row[pos] == raw_csv->opts.delimiter && 
				   first_row[pos + 1] == raw_csv->opts.delimiter) {
				++pos;
			}
			prev = pos + 1;
		}
	}
	return GDF_SUCCESS;
}

/**---------------------------------------------------------------------------*
 * @brief Updates the raw_csv_t object with the total number of rows and
 * quotation characters in the file
 *
 * Does not count the quotations if quotechar is set to '/0'.
 *
 * @param[in] h_data Pointer to the csv data in host memory
 * @param[in] h_size Size of the input data, in bytes
 * @param[in,out] raw_csv Structure containing the csv parsing parameters
 * and intermediate results
 *
 * @return gdf_error
 *---------------------------------------------------------------------------**/
gdf_error countRecordsAndQuotes(const char *h_data, size_t h_size, raw_csv_t *raw_csv) {
	vector<char> chars_to_count{raw_csv->opts.terminator};
	if (raw_csv->opts.quotechar != '\0') {
		chars_to_count.push_back(raw_csv->opts.quotechar);
	}

	raw_csv->num_records = countAllFromSet(h_data, h_size, chars_to_count);

	// If not starting at an offset, add an extra row to account for the first row in the file
	if (raw_csv->byte_range_offset == 0) {
		++raw_csv->num_records;
	}

	return GDF_SUCCESS;
}

/**---------------------------------------------------------------------------*
 * @brief Updates the raw_csv_t object with the offset of each row in the file
 * Also add positions of each quotation character in the file.
 *
 * Does not process the quotations if quotechar is set to '/0'.
 *
 * @param[in] h_data Pointer to the csv data in host memory
 * @param[in] h_size Size of the input data, in bytes
 * @param[in,out] raw_csv Structure containing the csv parsing parameters
 * and intermediate results
 *
 * @return gdf_error
 *---------------------------------------------------------------------------**/
gdf_error setRecordStarts(const char *h_data, size_t h_size, raw_csv_t *raw_csv) {
	// Allocate space to hold the record starting points
	const bool last_line_terminated = (h_data[h_size - 1] == raw_csv->opts.terminator);
	// If the last line is not terminated, allocate space for the EOF entry (added later)
	const gdf_size_type record_start_count = raw_csv->num_records + (last_line_terminated ? 0 : 1);
	raw_csv->recStart = device_buffer<uint64_t>(record_start_count); 

	auto* find_result_ptr = raw_csv->recStart.data();
	if (raw_csv->byte_range_offset == 0) {
		find_result_ptr++;
		CUDA_TRY(hipMemsetAsync(raw_csv->recStart.data(), 0ull, sizeof(uint64_t)));
	}
	vector<char> chars_to_find{raw_csv->opts.terminator};
	if (raw_csv->opts.quotechar != '\0') {
		chars_to_find.push_back(raw_csv->opts.quotechar);
	}
	// Passing offset = 1 to return positions AFTER the found character
	findAllFromSet(h_data, h_size, chars_to_find, 1, find_result_ptr);

	// Previous call stores the record pinput_file.typeositions as encountered by all threads
	// Sort the record positions as subsequent processing may require filtering
	// certain rows or other processing on specific records
	thrust::sort(rmm::exec_policy()->on(0), raw_csv->recStart.data(), raw_csv->recStart.data() + raw_csv->num_records);

	// Currently, ignoring lineterminations within quotes is handled by recording
	// the records of both, and then filtering out the records that is a quotechar
	// or a linetermination within a quotechar pair. The future major refactoring
	// of csv_reader and its kernels will probably use a different tactic.
	if (raw_csv->opts.quotechar != '\0') {
		vector<uint64_t> h_rec_starts(raw_csv->num_records);
		const size_t rec_start_size = sizeof(uint64_t) * (h_rec_starts.size());
		CUDA_TRY( hipMemcpy(h_rec_starts.data(), raw_csv->recStart.data(), rec_start_size, hipMemcpyDeviceToHost) );

		auto recCount = raw_csv->num_records;

		bool quotation = false;
		for (gdf_size_type i = 1; i < raw_csv->num_records; ++i) {
			if (h_data[h_rec_starts[i] - 1] == raw_csv->opts.quotechar) {
				quotation = !quotation;
				h_rec_starts[i] = raw_csv->num_bytes;
				recCount--;
			}
			else if (quotation) {
				h_rec_starts[i] = raw_csv->num_bytes;
				recCount--;
			}
		}

		CUDA_TRY( hipMemcpy(raw_csv->recStart.data(), h_rec_starts.data(), rec_start_size, hipMemcpyHostToDevice) );
		thrust::sort(rmm::exec_policy()->on(0), raw_csv->recStart.data(), raw_csv->recStart.data() + raw_csv->num_records);
		raw_csv->num_records = recCount;
	}

	if (!last_line_terminated){
		// Add the EOF as the last record when the terminator is missing in the last line
		const uint64_t eof_offset = h_size;
		CUDA_TRY(hipMemcpy(raw_csv->recStart.data() + raw_csv->num_records, &eof_offset, sizeof(uint64_t), hipMemcpyDefault));
		// Update the record count
		++raw_csv->num_records;
	}

	return GDF_SUCCESS;
}

/**---------------------------------------------------------------------------*
 * @brief Reads CSV-structured data and returns an array of gdf_columns.
 *
 * @param[in,out] args Structure containing input and output args
 *
 * @return gdf_error GDF_SUCCESS if successful, otherwise an error code.
 *---------------------------------------------------------------------------**/
gdf_error read_csv(csv_read_arg *args)
{
  gdf_error error = gdf_error::GDF_SUCCESS;

	//-----------------------------------------------------------------------------
	// create the CSV data structure - this will be filled in as the CSV data is processed.
	// Done first to validate data types
	raw_csv_t raw_csv{};
	// error = parseArguments(args, raw_csv);
	raw_csv.num_actual_cols	= args->num_cols;
	raw_csv.num_active_cols	= args->num_cols;
	raw_csv.num_records		= 0;

	raw_csv.header_row = args->header;
	raw_csv.skiprows = args->skiprows;
	raw_csv.skipfooter = args->skipfooter;
	raw_csv.nrows = args->nrows;
	raw_csv.prefix = args->prefix == nullptr ? "" : string(args->prefix);

	if (args->delim_whitespace) {
		raw_csv.opts.delimiter = ' ';
		raw_csv.opts.multi_delimiter = true;
	} else {
		raw_csv.opts.delimiter = args->delimiter;
		raw_csv.opts.multi_delimiter = false;
	}
	if (args->windowslinetermination) {
		raw_csv.opts.terminator = '\n';
	} else {
		raw_csv.opts.terminator = args->lineterminator;
	}
	if (args->quotechar != '\0' && args->quoting != QUOTE_NONE) {
		raw_csv.opts.quotechar = args->quotechar;
		raw_csv.opts.keepquotes = false;
		raw_csv.opts.doublequote = args->doublequote;
	} else {
		raw_csv.opts.quotechar = '\0';
		raw_csv.opts.keepquotes = true;
		raw_csv.opts.doublequote = false;
	}
	raw_csv.opts.skipblanklines = args->skip_blank_lines;
	raw_csv.opts.comment = args->comment;
	raw_csv.opts.dayfirst = args->dayfirst;
	raw_csv.opts.decimal = args->decimal;
	raw_csv.opts.thousands = args->thousands;
	if (raw_csv.opts.decimal == raw_csv.opts.delimiter) {
		checkError(GDF_INVALID_API_CALL, "Decimal point cannot be the same as the delimiter");
	}
	if (raw_csv.opts.thousands == raw_csv.opts.delimiter) {
		checkError(GDF_INVALID_API_CALL, "Thousands separator cannot be the same as the delimiter");
	}

	string compression_type;
	error = inferCompressionType(args->compression, args->filepath_or_buffer, compression_type);
	checkError(error, "call to inferCompressionType");

	raw_csv.byte_range_offset = args->byte_range_offset;
	raw_csv.byte_range_size = args->byte_range_size;
	if (raw_csv.byte_range_offset > 0 || raw_csv.byte_range_size > 0) {
		if (raw_csv.nrows >= 0 || raw_csv.skiprows > 0 || raw_csv.skipfooter > 0) {
			checkError(GDF_INVALID_API_CALL, 
				"Cannot manually limit rows to be read when using the byte range parameter");
		}
		if (compression_type != "none") {
			checkError(GDF_INVALID_API_CALL, 
				"Cannot read compressed input when using the byte range parameter");
		}
	}

	// Handle user-defined booleans values, whereby field data is substituted
	// with true/false values; CUDF booleans are int types of 0 or 1
	vector<string> true_values{"True", "TRUE"};
	if (args->true_values != nullptr && args->num_true_values > 0) {
		for (int i = 0; i < args->num_true_values; ++i) {
			true_values.emplace_back(args->true_values[i]);
		}
	}
	raw_csv.d_trueTrie = createSerializedTrie(true_values);
	raw_csv.opts.trueValuesTrie = raw_csv.d_trueTrie.data().get();

	vector<string> false_values{"False", "FALSE"};
	if (args->false_values != nullptr && args->num_false_values > 0) {
		for (int i = 0; i < args->num_false_values; ++i) {
			false_values.emplace_back(args->false_values[i]);
		}
	}
	raw_csv.d_falseTrie = createSerializedTrie(false_values);
	raw_csv.opts.falseValuesTrie = raw_csv.d_falseTrie.data().get();

	if (args->na_filter && 
		(args->keep_default_na || (args->na_values != nullptr && args->num_na_values > 0))) {
		vector<string> na_values{
			"#N/A", "#N/A N/A", "#NA", "-1.#IND", 
			"-1.#QNAN", "-NaN", "-nan", "1.#IND", 
			"1.#QNAN", "N/A", "NA", "NULL", 
			"NaN", "n/a", "nan", "null"};
		if(!args->keep_default_na){
			na_values.clear();
		}

		if (args->na_values != nullptr && args->num_na_values > 0) {
			for (int i = 0; i < args->num_na_values; ++i) {
				na_values.emplace_back(args->na_values[i]);
			}
		}

		raw_csv.d_naTrie = createSerializedTrie(na_values);
		raw_csv.opts.naValuesTrie = raw_csv.d_naTrie.data().get();
	}
	args->data = nullptr;

	//-----------------------------------------------------------------------------
	// memory map in the data
	void * 	map_data = NULL;
	size_t	map_size = 0;
	size_t	map_offset = 0;
	int fd = 0;
	if (args->input_data_form == gdf_csv_input_form::FILE_PATH)
	{
		fd = open(args->filepath_or_buffer, O_RDONLY );
		if (fd < 0) 		{ close(fd); checkError(GDF_FILE_ERROR, "Error opening file"); }

		struct stat st{};
		if (fstat(fd, &st)) { close(fd); checkError(GDF_FILE_ERROR, "cannot stat file");   }
	
		const auto file_size = st.st_size;
		const auto page_size = sysconf(_SC_PAGESIZE);

		if (args->byte_range_offset >= (size_t)file_size) { 
			close(fd); 
			checkError(GDF_INVALID_API_CALL, "The byte_range offset is larger than the file size");
		}

		// Have to align map offset to page size
		map_offset = (args->byte_range_offset/page_size)*page_size;

		// Set to rest-of-the-file size, will reduce based on the byte range size
		raw_csv.num_bytes = map_size = file_size - map_offset;

		// Include the page padding in the mapped size
		const size_t page_padding = args->byte_range_offset - map_offset;
		const size_t padded_byte_range_size = raw_csv.byte_range_size + page_padding;

		if (raw_csv.byte_range_size != 0 && padded_byte_range_size < map_size) {
			// Need to make sure that w/ padding we don't overshoot the end of file
			map_size = min(padded_byte_range_size + calculateMaxRowSize(args->num_cols), map_size);

		}

		// Ignore page padding for parsing purposes
		raw_csv.num_bytes = map_size - page_padding;

		map_data = mmap(0, map_size, PROT_READ, MAP_PRIVATE, fd, map_offset);
	
		if (map_data == MAP_FAILED || map_size==0) { close(fd); checkError(GDF_C_ERROR, "Error mapping file"); }
	}
	else if (args->input_data_form == gdf_csv_input_form::HOST_BUFFER)
	{
		map_data = (void *)args->filepath_or_buffer;
		raw_csv.num_bytes = map_size = args->buffer_size;
	}
	else { checkError(GDF_C_ERROR, "invalid input type"); }

	const char* h_uncomp_data;
	size_t h_uncomp_size = 0;
	// Used when the input data is compressed, to ensure the allocated uncompressed data is freed
	vector<char> h_uncomp_data_owner;
	if (compression_type == "none") {
		// Do not use the owner vector here to avoid copying the whole file to the heap
		h_uncomp_data = (const char*)map_data + (args->byte_range_offset - map_offset);
		h_uncomp_size = raw_csv.num_bytes;
	}
	else {
		error = getUncompressedHostData( (const char *)map_data, map_size, compression_type, h_uncomp_data_owner);
		checkError(error, "call to getUncompressedHostData");
		h_uncomp_data = h_uncomp_data_owner.data();
		h_uncomp_size = h_uncomp_data_owner.size();
	}
	assert(h_uncomp_data != nullptr);
	assert(h_uncomp_size != 0);

	error = countRecordsAndQuotes(h_uncomp_data, h_uncomp_size, &raw_csv);
	checkError(error, "call to count the number of rows");

	error = setRecordStarts(h_uncomp_data, h_uncomp_size, &raw_csv);
	checkError(error, "call to store the row offsets");

	error = uploadDataToDevice(h_uncomp_data, h_uncomp_size, &raw_csv);
	checkError(error, "call to upload the CSV data to the device");

	//-----------------------------------------------------------------------------
	//---  done with host data
	if (args->input_data_form == gdf_csv_input_form::FILE_PATH)
	{
		close(fd);
		munmap(map_data, map_size);
	}

	//-----------------------------------------------------------------------------
	//-- Populate the header

	// Check if the user gave us a list of column names
	if(args->names == nullptr) {

		error = setColumnNamesFromCsv(&raw_csv);
		if (error != GDF_SUCCESS) {
			return error;
		}
		const int h_num_cols = raw_csv.col_names.size();

		// Initialize a boolean array that states if a column needs to read or filtered.
		raw_csv.h_parseCol = thrust::host_vector<bool>(h_num_cols, true);
		
		// Rename empty column names to "Unnamed: col_index"
		for (size_t col_idx = 0; col_idx < raw_csv.col_names.size(); ++col_idx) {
			if (raw_csv.col_names[col_idx].empty()) {
				raw_csv.col_names[col_idx] = string("Unnamed: ") + std::to_string(col_idx);
			}
		}

		int h_dup_cols_removed = 0;
		// Looking for duplicates
		for (auto it = raw_csv.col_names.begin(); it != raw_csv.col_names.end(); it++){
			bool found_dupe = false;
			for (auto it2 = (it+1); it2 != raw_csv.col_names.end(); it2++){
				if (*it==*it2){
					found_dupe=true;
					break;
				}
			}
			if(found_dupe){
				int count=1;
				for (auto it2 = (it+1); it2 != raw_csv.col_names.end(); it2++){
					if (*it==*it2){
						if(args->mangle_dupe_cols){
							// Replace all the duplicates of column X with X.1,X.2,... First appearance stays as X.
							std::string newColName  = *it2;
							newColName += "." + std::to_string(count); 
							count++;
							*it2 = newColName;							
						} else{
							// All duplicate fields will be ignored.
							int pos=std::distance(raw_csv.col_names.begin(), it2);
							raw_csv.h_parseCol[pos]=false;
							h_dup_cols_removed++;
						}
					}
				}
			}
		}

		raw_csv.num_actual_cols = h_num_cols;							// Actual number of columns in the CSV file
		raw_csv.num_active_cols = h_num_cols-h_dup_cols_removed;		// Number of fields that need to be processed based on duplicatation fields

	}
	else {
		raw_csv.h_parseCol = thrust::host_vector<bool>(args->num_cols, true);

		for (int i = 0; i<raw_csv.num_actual_cols; i++){
			std::string col_name 	= args->names[i];
			raw_csv.col_names.push_back(col_name);
		}
	}

	// User can give
	if (args->use_cols_int!=NULL || args->use_cols_char!=NULL){
		if(args->use_cols_int!=NULL){
			for (int i = 0; i<raw_csv.num_actual_cols; i++)
				raw_csv.h_parseCol[i]=false;
			for(int i=0; i < args->use_cols_int_len; i++){
				int pos = args->use_cols_int[i];
				raw_csv.h_parseCol[pos]=true;
			}
			raw_csv.num_active_cols = args->use_cols_int_len;
		}else{
			for (int i = 0; i<raw_csv.num_actual_cols; i++)
				raw_csv.h_parseCol[i]=false;
			int countFound=0;
			for(int i=0; i < args->use_cols_char_len; i++){
				std::string colName(args->use_cols_char[i]);
				for (auto it = raw_csv.col_names.begin(); it != raw_csv.col_names.end(); it++){
					if(colName==*it){
						countFound++;
						int pos=std::distance(raw_csv.col_names.begin(), it);
						raw_csv.h_parseCol[pos]=true;
						break;
					}
				}
			}
			raw_csv.num_active_cols = countFound;
		}
	}
	raw_csv.d_parseCol = raw_csv.h_parseCol;

	//-----------------------------------------------------------------------------
	//---  done with host data
	if (args->input_data_form == gdf_csv_input_form::FILE_PATH)
	{
		close(fd);
		munmap(map_data, map_size);
	}


	//-----------------------------------------------------------------------------
	//--- Auto detect types of the vectors

	if(args->dtype==NULL){
		if (raw_csv.num_records == 0) {
			checkError(GDF_INVALID_API_CALL, "read_csv: no data available for data type inference");
		}

		vector<column_data_t> h_ColumnData(raw_csv.num_active_cols);
		device_buffer<column_data_t> d_ColumnData(raw_csv.num_active_cols);
		CUDA_TRY( hipMemset(d_ColumnData.data(),	0, 	(sizeof(column_data_t) * (raw_csv.num_active_cols)) ) ) ;

		launch_dataTypeDetection(&raw_csv, d_ColumnData.data());
		CUDA_TRY( hipMemcpy(h_ColumnData.data(), d_ColumnData.data(), sizeof(column_data_t) * (raw_csv.num_active_cols), hipMemcpyDeviceToHost));

		// host: array of dtypes (since gdf_columns are not created until end)
		vector<gdf_dtype>	d_detectedTypes;

		raw_csv.dtypes.clear();

		for(int col = 0; col < raw_csv.num_active_cols; col++){
			unsigned long long countInt = h_ColumnData[col].countInt8+h_ColumnData[col].countInt16+
										  h_ColumnData[col].countInt32+h_ColumnData[col].countInt64;

			if (h_ColumnData[col].countNULL == raw_csv.num_records){
				d_detectedTypes.push_back(GDF_INT8); // Entire column is NULL. Allocating the smallest amount of memory
			} else if(h_ColumnData[col].countString>0L){
				d_detectedTypes.push_back(GDF_STRING); // For auto-detection, we are currently not supporting strings.
			} else if(h_ColumnData[col].countDateAndTime>0L){
				d_detectedTypes.push_back(GDF_DATE64);
			} else if(h_ColumnData[col].countFloat > 0L  ||  
				(h_ColumnData[col].countFloat==0L && countInt >0L && h_ColumnData[col].countNULL >0L) ) {
				// The second condition has been added to conform to PANDAS which states that a colum of 
				// integers with a single NULL record need to be treated as floats.
				d_detectedTypes.push_back(GDF_FLOAT64);
			}
			else { 
				d_detectedTypes.push_back(GDF_INT64);
			}
		}
		raw_csv.dtypes=d_detectedTypes;
	}
	else{
		for ( int x = 0; x < raw_csv.num_actual_cols; x++) {

			std::string temp_type 	= args->dtype[x];
                        gdf_dtype col_dtype = GDF_invalid;
			if(temp_type.find(':') != std::string::npos){
				for (auto it = raw_csv.col_names.begin(); it != raw_csv.col_names.end(); it++){
				std::size_t idx = temp_type.find(':');
				if(temp_type.substr( 0, idx) == *it){
					std::string temp_dtype = temp_type.substr( idx +1);
					col_dtype	= convertStringToDtype(temp_dtype);
					break;
					}
				}
			}
			else{
				col_dtype	= convertStringToDtype( temp_type );
			}

			if (col_dtype == GDF_invalid)
				return GDF_UNSUPPORTED_DTYPE;

			raw_csv.dtypes.push_back(col_dtype);
		}
	}

  // Alloc output; columns' data memory is still expected for empty dataframe
  std::vector<gdf_column_wrapper> columns;
  for (int col = 0, active_col = 0; col < raw_csv.num_actual_cols; ++col) {
    if (raw_csv.h_parseCol[col]) {
      // When dtypes are inferred, it contains only active column values
      auto dtype = raw_csv.dtypes[args->dtype == nullptr ? active_col : col];

      columns.emplace_back(raw_csv.num_records, dtype,
                           gdf_dtype_extra_info{TIME_UNIT_NONE},
                           raw_csv.col_names[col]);
      CUDF_EXPECTS(columns.back().allocate() == GDF_SUCCESS, "Cannot allocate columns");
      active_col++;
    }
  }

  // Convert CSV input to cuDF output
  if (raw_csv.num_records != 0) {
    thrust::host_vector<gdf_dtype> h_dtypes(raw_csv.num_active_cols);
    thrust::host_vector<void*> h_data(raw_csv.num_active_cols);
    thrust::host_vector<gdf_valid_type*> h_valid(raw_csv.num_active_cols);

    for (int i = 0; i < raw_csv.num_active_cols; ++i) {
      h_dtypes[i] = columns[i]->dtype;
      h_data[i] = columns[i]->data;
      h_valid[i] = columns[i]->valid;
    }

    rmm::device_vector<gdf_dtype> d_dtypes = h_dtypes;
    rmm::device_vector<void*> d_data = h_data;
    rmm::device_vector<gdf_valid_type*> d_valid = h_valid;
    rmm::device_vector<gdf_size_type> d_valid_counts(raw_csv.num_active_cols, 0);

    CUDF_EXPECTS(
        launch_dataConvertColumns(&raw_csv, d_data.data().get(),
                                  d_valid.data().get(), d_dtypes.data().get(),
                                  d_valid_counts.data().get()) == GDF_SUCCESS,
        "Cannot convert CSV data to cuDF columns");
    CUDA_TRY(hipStreamSynchronize(0));

    thrust::host_vector<gdf_size_type> h_valid_counts = d_valid_counts;
    for (int i = 0; i < raw_csv.num_active_cols; ++i) {
      columns[i]->null_count = columns[i]->size - h_valid_counts[i];
    }
  }

  for (int i = 0; i < raw_csv.num_active_cols; ++i) {
    if (columns[i]->dtype == GDF_STRING) {
      std::unique_ptr<NVStrings, decltype(&NVStrings::destroy)> str_data(
        NVStrings::create_from_index(static_cast<string_pair *>(columns[i]->data), columns[i]->size), 
        &NVStrings::destroy);
      RMM_TRY(RMM_FREE(columns[i]->data, 0));

      // PANDAS' default behavior of enabling doublequote for two consecutive
      // quotechars in quoted fields results in reduction to a single quotechar
      if ((raw_csv.opts.quotechar != '\0') &&
          (raw_csv.opts.doublequote == true)) {
        const std::string quotechar(1, raw_csv.opts.quotechar);
        const std::string doublequotechar(2, raw_csv.opts.quotechar);
        columns[i]->data = str_data->replace(doublequotechar.c_str(), quotechar.c_str());
      }
      else {
        columns[i]->data = str_data.release();
      }
    }
  }

  // Transfer ownership to raw pointer output arguments
  args->data = (gdf_column **)malloc(sizeof(gdf_column *) * raw_csv.num_active_cols);
  for (int i = 0; i < raw_csv.num_active_cols; ++i) {
    args->data[i] = columns[i].release();
  }
  args->num_cols_out = raw_csv.num_active_cols;
  args->num_rows_out = raw_csv.num_records;

  return error;
}

/**---------------------------------------------------------------------------*
 * @brief Infer the compression type from the compression parameter and 
 * the input file name
 * 
 * Returns "none" if the input is not compressed.
 * 
 * @param[in] compression_arg Input string that is potentially describing 
 * the compression type. Can also be nullptr, "none", or "infer"
 * @param[in] filepath path + name of the input file
 * @param[out] compression_type String describing the inferred compression type
 * 
 * @return gdf_error with error code on failure, otherwise GDF_SUCCESS
 *---------------------------------------------------------------------------**/
gdf_error inferCompressionType(const char* compression_arg, const char* filepath, string& compression_type)
{
	if (compression_arg && 0 == strcasecmp(compression_arg, "none")) {
		compression_arg = nullptr;
	}
	if (compression_arg && 0 == strcasecmp(compression_arg, "infer"))
	{
		const char *file_ext = strrchr(filepath, '.');
		compression_arg = nullptr;
		if (file_ext)
		{
			if (!strcasecmp(file_ext, ".gz"))
				compression_arg = "gzip";
			else if (!strcasecmp(file_ext, ".zip"))
				compression_arg = "zip";
			else if (!strcasecmp(file_ext, ".bz2"))
				compression_arg = "bz2";
			else if (!strcasecmp(file_ext, ".xz"))
				compression_arg = "xz";
			else {
				// TODO: return error here
			}
		}
	}
	compression_type = compression_arg == nullptr? "none":string(compression_arg);
	
	return GDF_SUCCESS;
}

/**---------------------------------------------------------------------------*
 * @brief Uploads the relevant segment of the input csv data onto the GPU.
 * 
 * Only rows that need to be read are copied to the GPU, based on parameters
 * like nrows, skipheader, skipfooter.
 * Also updates the array of record starts to match the device data offset.
 * 
 * @param[in] h_uncomp_data Pointer to the uncompressed csv data in host memory
 * @param[in] h_uncomp_size Size of the input data, in bytes
 * @param[in,out] raw_csv Structure containing the csv parsing parameters
 * and intermediate results
 * 
 * @return gdf_error with error code on failure, otherwise GDF_SUCCESS
 *---------------------------------------------------------------------------**/
gdf_error uploadDataToDevice(const char *h_uncomp_data, size_t h_uncomp_size,
                             raw_csv_t *raw_csv) {

  // Exclude the rows that are to be skipped from the start
  GDF_REQUIRE(raw_csv->num_records > raw_csv->skiprows, GDF_INVALID_API_CALL);
  const auto first_row = raw_csv->skiprows;
  raw_csv->num_records = raw_csv->num_records - first_row;

  std::vector<uint64_t> h_rec_starts(raw_csv->num_records);
  CUDA_TRY(hipMemcpy(h_rec_starts.data(), raw_csv->recStart.data() + first_row,
                      sizeof(uint64_t) * h_rec_starts.size(),
                      hipMemcpyDefault));

  // Trim lines that are outside range, but keep one greater for the end offset
  if (raw_csv->byte_range_size != 0) {
    auto it = h_rec_starts.end() - 1;
    while (it >= h_rec_starts.begin() &&
           *it > uint64_t(raw_csv->byte_range_size)) {
      --it;
    }
    if ((it + 2) < h_rec_starts.end()) {
      h_rec_starts.erase(it + 2, h_rec_starts.end());
    }
  }

  // Discard only blank lines, only fully comment lines, or both.
  // If only handling one of them, ensure it doesn't match against \0 as we do
  // not want certain scenarios to be filtered out (end-of-file)
  if (raw_csv->opts.skipblanklines || raw_csv->opts.comment != '\0') {
    const auto match_newline = raw_csv->opts.skipblanklines ? raw_csv->opts.terminator
                                                            : raw_csv->opts.comment;
    const auto match_comment = raw_csv->opts.comment != '\0' ? raw_csv->opts.comment
                                                             : match_newline;
    const auto match_return = (raw_csv->opts.skipblanklines &&
                              raw_csv->opts.terminator == '\n') ? '\r'
                                                                : match_comment;
    h_rec_starts.erase(
        std::remove_if(h_rec_starts.begin(), h_rec_starts.end(),
                       [&](uint64_t i) {
                         return (h_uncomp_data[i] == match_newline ||
                                 h_uncomp_data[i] == match_return ||
                                 h_uncomp_data[i] == match_comment);
                       }),
        h_rec_starts.end());
  }

  raw_csv->num_records = h_rec_starts.size();

  // Exclude the rows before the header row (inclusive)
  // But copy the header data for parsing the column names later (if necessary)
  if (raw_csv->header_row >= 0) {
    raw_csv->header.assign(
        h_uncomp_data + h_rec_starts[raw_csv->header_row],
        h_uncomp_data + h_rec_starts[raw_csv->header_row + 1]);
    h_rec_starts.erase(h_rec_starts.begin(),
                       h_rec_starts.begin() + raw_csv->header_row + 1);
    raw_csv->num_records = h_rec_starts.size();
  }

  // Exclude the rows that exceed past the requested number
  if (raw_csv->nrows >= 0 && raw_csv->nrows < raw_csv->num_records) {
    h_rec_starts.resize(raw_csv->nrows + 1);    // include end offset
    raw_csv->num_records = h_rec_starts.size();
  }

  // Exclude the rows that are to be skipped from the end
  if (raw_csv->skipfooter > 0) {
    h_rec_starts.resize(h_rec_starts.size() - raw_csv->skipfooter);
    raw_csv->num_records = h_rec_starts.size();
  }

  // Check that there is actual data to parse
  GDF_REQUIRE(raw_csv->num_records > 0, GDF_INVALID_API_CALL);

  const auto start_offset = h_rec_starts.front();
  const auto end_offset = h_rec_starts.back();
  raw_csv->num_bytes = end_offset - start_offset;
  assert(raw_csv->num_bytes <= h_uncomp_size);
  raw_csv->num_bits = (raw_csv->num_bytes + 63) / 64;

  // Resize and upload the rows of interest
  raw_csv->recStart.resize(raw_csv->num_records);
  CUDA_TRY(hipMemcpy(raw_csv->recStart.data(), h_rec_starts.data(),
                      sizeof(uint64_t) * raw_csv->num_records,
                      hipMemcpyDefault));

  // Upload the raw data that is within the rows of interest
  raw_csv->data = device_buffer<char>(raw_csv->num_bytes);
  CUDA_TRY(hipMemcpy(raw_csv->data.data(), h_uncomp_data + start_offset,
                      raw_csv->num_bytes, hipMemcpyHostToDevice));

  // Adjust row start positions to account for the data subcopy
  thrust::transform(rmm::exec_policy()->on(0), raw_csv->recStart.data(),
                    raw_csv->recStart.data() + raw_csv->num_records,
                    thrust::make_constant_iterator(start_offset),
                    raw_csv->recStart.data(), thrust::minus<uint64_t>());

  // The array of row offsets includes EOF
  // reduce the number of records by one to exclude it from the row count
  raw_csv->num_records--;

  return GDF_SUCCESS;
}

//----------------------------------------------------------------------------------------------------------------
//				CUDA Kernels
//----------------------------------------------------------------------------------------------------------------

/**---------------------------------------------------------------------------*
 * @brief Helper function to setup and launch CSV parsing CUDA kernel.
 * 
 * @param[in,out] raw_csv The metadata for the CSV data
 * @param[out] gdf The output column data
 * @param[out] valid The bitmaps indicating whether column fields are valid
 * @param[out] str_cols The start/end offsets for string data types
 * @param[out] num_valid The numbers of valid fields in columns
 *
 * @return gdf_error GDF_SUCCESS upon completion
 *---------------------------------------------------------------------------**/
gdf_error launch_dataConvertColumns(raw_csv_t *raw_csv, void **gdf,
                                    gdf_valid_type **valid, gdf_dtype *d_dtypes,
                                    gdf_size_type *num_valid) {
  int blockSize;    // suggested thread count to use
  int minGridSize;  // minimum block count required
  CUDA_TRY(hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize,
                                              convertCsvToGdf));

  // Calculate actual block count to use based on records count
  int gridSize = (raw_csv->num_records + blockSize - 1) / blockSize;

  convertCsvToGdf <<< gridSize, blockSize >>> (
      raw_csv->data.data(), raw_csv->opts, raw_csv->num_records,
      raw_csv->num_actual_cols, raw_csv->d_parseCol.data().get(), raw_csv->recStart.data(),
      d_dtypes, gdf, valid, num_valid);

  CUDA_TRY(hipGetLastError());
  return GDF_SUCCESS;
}

/**---------------------------------------------------------------------------*
 * @brief Functor for converting CSV data to cuDF data type value.
 *---------------------------------------------------------------------------**/
struct ConvertFunctor {
  /**---------------------------------------------------------------------------*
   * @brief Template specialization for operator() for types whose values can be
   * convertible to a 0 or 1 to represent false/true. The converting is done by
   * checking against the default and user-specified true/false values list.
   *
   * It is handled here rather than within convertStrToValue() as that function
   * is used by other types (ex. timestamp) that aren't 'booleable'.
   *---------------------------------------------------------------------------**/
  template <typename T,
            typename std::enable_if_t<std::is_integral<T>::value> * = nullptr>
  __host__ __device__ __forceinline__ void operator()(
      const char *csvData, void *gdfColumnData, long rowIndex, long start,
      long end, const ParseOptions &opts) {
    T &value{static_cast<T *>(gdfColumnData)[rowIndex]};

    // Check for user-specified true/false values first, where the output is
    // replaced with 1/0 respectively
    const size_t field_len = end - start + 1;
    if (serializedTrieContains(opts.trueValuesTrie, csvData + start, field_len)) {
      value = 1;
    } else if (serializedTrieContains(opts.falseValuesTrie, csvData + start, field_len)) {
      value = 0;
    } else {
      value = convertStrToValue<T>(csvData, start, end, opts);
    }
  }

  /**---------------------------------------------------------------------------*
   * @brief Default template operator() dispatch specialization all data types
   * (including wrapper types) that is not covered by above.
   *---------------------------------------------------------------------------**/
  template <typename T,
            typename std::enable_if_t<!std::is_integral<T>::value> * = nullptr>
  __host__ __device__ __forceinline__ void operator()(
      const char *csvData, void *gdfColumnData, long rowIndex, long start,
      long end, const ParseOptions &opts) {
    T &value{static_cast<T *>(gdfColumnData)[rowIndex]};
    value = convertStrToValue<T>(csvData, start, end, opts);
  }
};

/**---------------------------------------------------------------------------*
 * @brief CUDA kernel that parses and converts CSV data into cuDF column data.
 * 
 * Data is processed one record at a time
 *
 * @param[in] raw_csv The entire CSV data to read
 * @param[in] opts A set of parsing options
 * @param[in] num_records The number of lines/rows of CSV data
 * @param[in] num_columns The number of columns of CSV data
 * @param[in] parseCol Whether to parse or skip a column
 * @param[in] recStart The start the CSV data of interest
 * @param[in] dtype The data type of the column
 * @param[out] gdf_data The output column data
 * @param[out] valid The bitmaps indicating whether column fields are valid
 * @param[out] num_valid The numbers of valid fields in columns
 *
 * @return gdf_error GDF_SUCCESS upon completion
 *---------------------------------------------------------------------------**/
__global__ void convertCsvToGdf(char *raw_csv, const ParseOptions opts,
                                gdf_size_type num_records, int num_columns,
                                bool *parseCol, uint64_t *recStart,
                                gdf_dtype *dtype, void **gdf_data,
                                gdf_valid_type **valid,
                                gdf_size_type *num_valid)
{
	// thread IDs range per block, so also need the block id
	long	rec_id  = threadIdx.x + (blockDim.x * blockIdx.x);		// this is entry into the field array - tid is an elements within the num_entries array

	// we can have more threads than data, make sure we are not past the end of the data
	if ( rec_id >= num_records)
		return;

	long start 		= recStart[rec_id];
	long stop 		= recStart[rec_id + 1];

	long pos 		= start;
	int  col 		= 0;
	int  actual_col = 0;

	while(col<num_columns){

		if(start>stop)
			break;

		pos = seekFieldEnd(raw_csv, opts, pos, stop);

		if(parseCol[col]==true){

			// check if the entire field is a NaN string - consistent with pandas
			const bool is_na = serializedTrieContains(opts.naValuesTrie, raw_csv + start, pos - start);

			// Modify start & end to ignore whitespace and quotechars
			long tempPos=pos-1;
			if(!is_na && dtype[actual_col] != gdf_dtype::GDF_CATEGORY && dtype[actual_col] != gdf_dtype::GDF_STRING){
				adjustForWhitespaceAndQuotes(raw_csv, &start, &tempPos, opts.quotechar);
			}

			if(!is_na && start<=(tempPos)) { // Empty fields are not legal values

				// Type dispatcher does not handle GDF_STRINGS
				if (dtype[actual_col] == gdf_dtype::GDF_STRING) {
					long end = pos;
					if(opts.keepquotes==false){
						if((raw_csv[start] == opts.quotechar) && (raw_csv[end-1] == opts.quotechar)){
							start++;
							end--;
						}
					}
					auto str_list = static_cast<string_pair*>(gdf_data[actual_col]);
					str_list[rec_id].first = raw_csv + start;
					str_list[rec_id].second = end - start;
				} else {
					cudf::type_dispatcher(
						dtype[actual_col], ConvertFunctor{}, raw_csv,
						gdf_data[actual_col], rec_id, start, tempPos, opts);
				}

				// set the valid bitmap - all bits were set to 0 to start
				setBitmapBit(valid[actual_col], rec_id);
				atomicAdd(&num_valid[actual_col], 1);
			}
			else if(dtype[actual_col]==gdf_dtype::GDF_STRING){
				auto str_list = static_cast<string_pair*>(gdf_data[actual_col]);
				str_list[rec_id].first = nullptr;
				str_list[rec_id].second = 0;
			}
			actual_col++;
		}
		pos++;
		start=pos;
		col++;

	}
}

/**---------------------------------------------------------------------------*
 * @brief Helper function to setup and launch CSV data type detect CUDA kernel.
 * 
 * @param[in] raw_csv The metadata for the CSV data
 * @param[out] d_columnData The count for each column data type
 *
 * @return gdf_error GDF_SUCCESS upon completion
 *---------------------------------------------------------------------------**/
gdf_error launch_dataTypeDetection(raw_csv_t *raw_csv,
                                   column_data_t *d_columnData) {
  int blockSize;    // suggested thread count to use
  int minGridSize;  // minimum block count required
  CUDA_TRY(hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize,
                                              dataTypeDetection));

  // Calculate actual block count to use based on records count
  int gridSize = (raw_csv->num_records + blockSize - 1) / blockSize;

  dataTypeDetection <<< gridSize, blockSize >>> (
      raw_csv->data.data(), raw_csv->opts, raw_csv->num_records,
      raw_csv->num_actual_cols, raw_csv->d_parseCol.data().get(), raw_csv->recStart.data(),
      d_columnData);

  CUDA_TRY(hipGetLastError());
  return GDF_SUCCESS;
}

/**---------------------------------------------------------------------------*
 * @brief CUDA kernel that parses and converts CSV data into cuDF column data.
 *
 * Data is processed in one row/record at a time, so the number of total
 * threads (tid) is equal to the number of rows.
 *
 * @param[in] raw_csv The entire CSV data to read
 * @param[in] opts A set of parsing options
 * @param[in] num_records The number of lines/rows of CSV data
 * @param[in] num_columns The number of columns of CSV data
 * @param[in] parseCol Whether to parse or skip a column
 * @param[in] recStart The start the CSV data of interest
 * @param[out] d_columnData The count for each column data type
 *
 * @returns GDF_SUCCESS upon successful computation
 *---------------------------------------------------------------------------**/
__global__
void dataTypeDetection(char *raw_csv,
                       const ParseOptions opts,
                       gdf_size_type num_records,
                       int num_columns,
                       bool *parseCol,
                       uint64_t *recStart,
                       column_data_t *d_columnData)
{
	// thread IDs range per block, so also need the block id
	long	rec_id  = threadIdx.x + (blockDim.x * blockIdx.x);		// this is entry into the field array - tid is an elements within the num_entries array

	// we can have more threads than data, make sure we are not past the end of the data
	if ( rec_id >= num_records)
		return;

	long start 		= recStart[rec_id];
	long stop 		= recStart[rec_id + 1];

	long pos 		= start;
	int  col 		= 0;
	int  actual_col = 0;

	// Going through all the columns of a given record
	while(col<num_columns){

		if(start>stop)
			break;

		pos = seekFieldEnd(raw_csv, opts, pos, stop);

		// Checking if this is a column that the user wants --- user can filter columns
		if(parseCol[col]==true){

			long tempPos=pos-1;

			// Checking if the record is NULL
			if(start>(tempPos)){
				atomicAdd(& d_columnData[actual_col].countNULL, 1L);
				pos++;
				start=pos;
				col++;
				actual_col++;
				continue;	
			}

			long countNumber=0;
			long countDecimal=0;
			long countSlash=0;
			long countDash=0;
			long countColon=0;
			long countString=0;
			long countExponent=0;

			// Modify start & end to ignore whitespace and quotechars
			// This could possibly result in additional empty fields
			adjustForWhitespaceAndQuotes(raw_csv, &start, &tempPos);

			const long strLen = tempPos - start + 1;

			const bool maybe_hex = ((strLen > 2 && raw_csv[start] == '0' && raw_csv[start + 1] == 'x') ||
				(strLen > 3 && raw_csv[start] == '-' && raw_csv[start + 1] == '0' && raw_csv[start + 2] == 'x'));

			for(long startPos=start; startPos<=tempPos; startPos++){
				if(isDigit(raw_csv[startPos], maybe_hex)){
					countNumber++;
					continue;
				}
				// Looking for unique characters that will help identify column types.
				switch (raw_csv[startPos]){
					case '.':
						countDecimal++;break;
					case '-':
						countDash++; break;
					case '/':
						countSlash++;break;
					case ':':
						countColon++;break;
					case 'e':
					case 'E':
						if (!maybe_hex && startPos > start && startPos < tempPos) 
							countExponent++;break;
					default:
						countString++;
						break;	
				}
			}

			// Integers have to have the length of the string
			long int_req_number_cnt = strLen;
			// Off by one if they start with a minus sign
			if(raw_csv[start]=='-' && strLen > 1){
				--int_req_number_cnt;
			}
			// Off by one if they are a hexadecimal number
			if(maybe_hex) {
				--int_req_number_cnt;
			}

			if(strLen==0){ // Removed spaces ' ' in the pre-processing and thus we can have an empty string.
				atomicAdd(& d_columnData[actual_col].countNULL, 1L);
			}
			else if(countNumber==int_req_number_cnt){
				// Checking to see if we the integer value requires 8,16,32,64 bits.
				// This will allow us to allocate the exact amount of memory.
				const auto value = convertStrToValue<int64_t>(raw_csv, start, tempPos, opts);
				const size_t field_len = tempPos - start + 1;
				if (serializedTrieContains(opts.trueValuesTrie, raw_csv + start, field_len) ||
					serializedTrieContains(opts.falseValuesTrie, raw_csv + start, field_len)){
					atomicAdd(& d_columnData[actual_col].countInt8, 1L);
				}
				else if(value >= (1L<<31)){
					atomicAdd(& d_columnData[actual_col].countInt64, 1L);
				}
				else if(value >= (1L<<15)){
					atomicAdd(& d_columnData[actual_col].countInt32, 1L);
				}
				else if(value >= (1L<<7)){
					atomicAdd(& d_columnData[actual_col].countInt16, 1L);
				}
				else{
					atomicAdd(& d_columnData[actual_col].countInt8, 1L);
				}
			}
			else if(isLikeFloat(strLen, countNumber, countDecimal, countDash, countExponent)){
					atomicAdd(& d_columnData[actual_col].countFloat, 1L);
			}
			// The date-time field cannot have more than 3 strings. As such if an entry has more than 3 string characters, it is not 
			// a data-time field. Also, if a string has multiple decimals, then is not a legit number.
			else if(countString > 3 || countDecimal > 1){
				atomicAdd(& d_columnData[actual_col].countString, 1L);
			}
			else {
				// A date field can have either one or two '-' or '\'. A legal combination will only have one of them.
				// To simplify the process of auto column detection, we are not covering all the date-time formation permutations.
				if((countDash>0 && countDash<=2 && countSlash==0)|| (countDash==0 && countSlash>0 && 	countSlash<=2) ){
					if((countColon<=2)){
						atomicAdd(& d_columnData[actual_col].countDateAndTime, 1L);
					}
					else{
						atomicAdd(& d_columnData[actual_col].countString, 1L);					
					}
				}
				// Default field is string type.
				else{
					atomicAdd(& d_columnData[actual_col].countString, 1L);					
				}
			}
			actual_col++;
		}
		pos++;
		start=pos;
		col++;	

	}
}
