#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2018, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
	 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

/**
 * @file csv-reader.cu  code to read csv data
 *
 * CSV Reader
 */


#include <hip/hip_runtime.h>

#include <iostream>
#include <vector>
#include <string>
#include <stdio.h>
#include <iostream>
#include <iomanip>
#include <vector>
#include <unordered_map>

#include <stdio.h>
#include <stdlib.h>

#include <unistd.h>
#include <fcntl.h>
#include <sys/types.h>
#include <sys/stat.h>
#include <sys/mman.h>

#include <thrust/scan.h>
#include <thrust/reduce.h>
#include <thrust/device_ptr.h>
#include <thrust/execution_policy.h>

#include <thrust/host_vector.h>

#include "type_conversion.cuh"
#include "datetime_parser.cuh"

#include "cudf.h"
#include "utilities/error_utils.h"
 
#include "rmm/rmm.h"

#include "NVStrings.h"

constexpr int32_t HASH_SEED = 33;

using namespace std;

//-- define the structure for raw data handling - for internal use
typedef struct raw_csv_ {
    char *				data;			// on-device: the raw unprocessed CSV data - loaded as a large char * array
    unsigned long long*	d_num_records;	// on-device: Number of records.
    unsigned long long*	recStart;		// on-device: Starting position of the records.

    char				delimiter;		// host: the delimiter
    char				terminator;		// host: the line terminator

    char				quotechar;		// host: the quote character
    bool				keepquotes;		// host: indicates to keep the start and end quotechar
    bool				doublequote;	// host: indicates to interpret two consecutive quotechar as a single

    long				num_bytes;		// host: the number of bytes in the data
    long				num_bits;		// host: the number of 64-bit bitmaps (different than valid)
	unsigned long long 	num_records;  	// host: number of records (per column)
	// int				num_cols;		// host: number of columns
	int					num_active_cols;	// host: number of columns that will be return to user.
	int					num_actual_cols;	// host: number of columns in the file --- based on the number of columns in header
    vector<gdf_dtype>	dtypes;			// host: array of dtypes (since gdf_columns are not created until end)
    vector<string>		col_names;		// host: array of column names
    bool* 				h_parseCol;		// host   : array of booleans stating if column should be parsed in reading process: parseCol[x]=false means that the column x needs to be filtered out.
    bool* 				d_parseCol;		// device : array of booleans stating if column should be parsed in reading process: parseCol[x]=false means that the column x needs to be filtered out.
    long 				header_row;		// Row id of the header
    bool				dayfirst;
    char				decimal;
    char				thousands;
} raw_csv_t;

typedef struct column_data_ {
	unsigned long long countFloat;
	unsigned long long countDateAndTime;
	unsigned long long countString;
	unsigned long long countInt8;
	unsigned long long countInt16;
	unsigned long long countInt32;
	unsigned long long countInt64;	
	unsigned long long countNULL;
} column_data_t;

typedef struct parsing_opts_ {
	char				delimiter;
	char				terminator;
	char				quotechar;
	bool				keepquotes;
	char				decimal;
	char				thousands;
} parsing_opts_t;

using string_pair = std::pair<const char*,size_t>;

//
//---------------create and process ---------------------------------------------
//
gdf_error parseArguments(csv_read_arg *args, raw_csv_t *csv);
// gdf_error getColNamesAndTypes(const char **col_names, const  char **dtypes, raw_csv_t *d);
gdf_error updateRawCsv( const char * data, long num_bytes, raw_csv_t * csvData );
gdf_error allocateGdfDataSpace(gdf_column *);
gdf_dtype convertStringToDtype(std::string &dtype);

#define checkError(error, txt)  if ( error != GDF_SUCCESS) { cerr << "ERROR:  " << error <<  "  in "  << txt << endl;  return error; }

//
//---------------CUDA Kernel ---------------------------------------------
//

__device__ int findSetBit(int tid, long num_bits, uint64_t *f_bits, int x);

gdf_error launch_countRecords(raw_csv_t * csvData);
gdf_error launch_storeRecordStart(raw_csv_t * csvData);
gdf_error launch_dataConvertColumns(raw_csv_t * raw_csv, void** d_gdf,  gdf_valid_type** valid, gdf_dtype* d_dtypes, string_pair	**str_cols, long row_offset, unsigned long long *);

gdf_error launch_dataTypeDetection(raw_csv_t * raw_csv, long row_offset, column_data_t* d_columnData);

__global__ void countRecords(char *data, const char terminator, const char quotechar, long num_bytes, long num_bits, unsigned long long* num_records);
__global__ void storeRecordStart(char *data, const char terminator, const char quotechar, long num_bytes, long num_bits, unsigned long long* num_records,unsigned long long* recStart) ;
__global__ void convertCsvToGdf(char *csv, const parsing_opts_t opts, unsigned long long num_records, int num_columns,bool *parseCol,unsigned long long *recStart,gdf_dtype *dtype,void **gdf_data,gdf_valid_type **valid,string_pair **str_cols,unsigned long long row_offset, long header_row,bool dayfirst,unsigned long long *num_valid);
__global__ void dataTypeDetection(char *raw_csv, const parsing_opts_t opts, unsigned long long num_records, int  num_columns, bool  *parseCol, unsigned long long *recStart, unsigned long long row_offset, long header_row, column_data_t* d_columnData);

//
//---------------CUDA Valid (8 blocks of 8-bits) Bitmap Kernels ---------------------------------------------
//
__device__ int whichBitmap(int record) { return (record/8);  }
__device__ int whichBit(int bit) { return (bit % 8);  }

__inline__ __device__ void validAtomicOR(gdf_valid_type* address, gdf_valid_type val)
{
	int32_t *base_address = (int32_t*)((gdf_valid_type*)address - ((size_t)address & 3));
	int32_t int_val = (int32_t)val << (((size_t) address & 3) * 8);

	atomicOr(base_address, int_val);
}

__device__ void setBit(gdf_valid_type* address, int bit) {
	gdf_valid_type bitMask[8] 		= {1, 2, 4, 8, 16, 32, 64, 128};
	validAtomicOR(address, bitMask[bit]);
}




std::string stringType(gdf_dtype dt){

	switch (dt){
		case GDF_STRING: return std::string("str");
		case GDF_DATE64: return std::string("date64");
		case GDF_CATEGORY: return std::string("category");
		case GDF_FLOAT64: return std::string("float64");
		case GDF_INT8: return std::string("int8");
		case GDF_INT16: return std::string("int16");
		case GDF_INT32: return std::string("int32");
		case GDF_INT64: return std::string("int64");
		default:
			return "long";
	}


}



/**
 * @brief read in a CSV file
 *
 * Read in a CSV file, extract all fields, and return a GDF (array of gdf_columns)
 *
 * @param[in and out] args the input arguments, but this also contains the returned data
 *
 * Arguments:
 *
 *  Required Arguments
 * 		file_path			-	file location to read from	- currently the file cannot be compressed
 * 		num_cols			-	number of columns in the names and dtype arrays
 * 		names				-	ordered List of column names, this is a required field
 * 		dtype				-	ordered List of data types, this is required
 *
 * 	Optional
 * 		lineterminator		-	define the line terminator character.  Default is '\n'
 * 		delimiter			-	define the field separator, default is ','.  This argument is also called 'sep'
 *
 * 		quotechar;				define the character used to denote start and end of a quoted item
 * 		quoting;				treat string fields as quoted item and remove the first and last quotechar
 * 		nodoublequote;			do not interpret two consecutive quotechar as a single quotechar
 *
 * 		delim_whitespace	-	use white space as the delimiter - default is false.  This overrides the delimiter argument
 * 		skipinitialspace	-	skip white spaces after the delimiter - default is false
 *
 * 		skiprows			-	number of rows at the start of the files to skip, default is 0
 * 		skipfooter			-	number of rows at the bottom of the file to skip - default is 0
 *
 * 		dayfirst			-	is the first value the day?  DD/MM  versus MM/DD
 *
 *
 *  Output
 *  	num_cols_out		-	Out: return the number of columns read in
 *  	num_rows_out		-	Out: return the number of rows read in
 *  	gdf_column **data	-	Out: return the array of *gdf_columns
 *
 *
 * @return gdf_error
 *
 */
gdf_error read_csv(csv_read_arg *args)
{
	gdf_error error = gdf_error::GDF_SUCCESS;

	//-----------------------------------------------------------------------------
	// create the CSV data structure - this will be filled in as the CSV data is processed.
	// Done first to validate data types
	raw_csv_t * raw_csv = new raw_csv_t;
	// error = parseArguments(args, raw_csv);
	raw_csv->num_actual_cols	= args->num_cols;
	raw_csv->num_active_cols	= args->num_cols;
	raw_csv->num_records		= 0;

	if(args->delim_whitespace == true) {
		raw_csv->delimiter = ' ';
	} else {
		raw_csv->delimiter = args->delimiter;
	}

	if(args->windowslinetermination) {
		raw_csv->terminator = '\n';
	} else {
		raw_csv->terminator = args->lineterminator;
	}

	raw_csv->quotechar = args->quotechar;
	if(raw_csv->quotechar != '\0') {
		raw_csv->keepquotes = !args->quoting;
		raw_csv->doublequote = args->doublequote;
	} else {
		raw_csv->keepquotes = true;
		raw_csv->doublequote = false;
	}

	raw_csv->dayfirst = args->dayfirst;
	raw_csv->decimal = args->decimal;
	raw_csv->thousands = args->thousands == nullptr ? '\0' : *args->thousands;

	if (raw_csv->decimal == raw_csv->delimiter)
	{ 
		checkError(GDF_INVALID_API_CALL, "Decimal point cannot be the same as the delimiter");
	}
	if (raw_csv->thousands == raw_csv->delimiter)
	{ 
		checkError(GDF_INVALID_API_CALL, "Thousands separator cannot be the same as the delimiter");
	}

	//-----------------------------------------------------------------------------
	// memory map in the data
	void * 			map_data = NULL;
	struct stat     st;
	int				fd;

	fd = open(args->file_path, O_RDONLY );

	if (fd < 0) 		{ close(fd); checkError(GDF_FILE_ERROR, "Error opening file"); }
	if (fstat(fd, &st)) { close(fd); checkError(GDF_FILE_ERROR, "cannot stat file");   }

	raw_csv->num_bytes = st.st_size;

	map_data = mmap(0, raw_csv->num_bytes, PROT_READ, MAP_PRIVATE, fd, 0);

    if (map_data == MAP_FAILED || raw_csv->num_bytes==0) { close(fd); checkError(GDF_C_ERROR, "Error mapping file"); }

	//-----------------------------------------------------------------------------
	//---  create a structure to hold variables used to parse the CSV data
	error = updateRawCsv( (const char *)map_data, (long)raw_csv->num_bytes, raw_csv );
	checkError(error, "call to createRawCsv");

	//-----------------------------------------------------------------------------
	// find the record and fields points (in bitmaps)
	error = launch_countRecords(raw_csv);
	checkError(error, "call to record counter");

	//-----------------------------------------------------------------------------
	//-- Allocate space to hold the record starting point
	RMM_TRY( RMM_ALLOC((void**)&(raw_csv->recStart), (sizeof(unsigned long long) * (raw_csv->num_records + 1)), 0) ); 
	CUDA_TRY( hipMemset(raw_csv->d_num_records,	0, 		(sizeof(unsigned long long) )) ) ;

	//-----------------------------------------------------------------------------
	//-- Scan data and set the starting positions
	error = launch_storeRecordStart(raw_csv);
	checkError(error, "call to record initial position store");

	// Previous kernel stores the record positions as encountered by all threads
	// Sort the record positions as subsequent processing may require filtering
	// certain rows or other processing on specific records
	thrust::sort(thrust::device, raw_csv->recStart, raw_csv->recStart + raw_csv->num_records + 1);

	// Currently, ignoring lineterminations within quotes is handled by recording
	// the records of both, and then filtering out the records that is a quotechar
	// or a linetermination within a quotechar pair. The future major refactoring
	// of csv_reader and its kernels will probably use a different tactic.
	if (raw_csv->quotechar != '\0') {
		const size_t recTotalSize = sizeof(unsigned long long) * (raw_csv->num_records + 1);

		unsigned long long *h_recStart = (unsigned long long*)malloc(recTotalSize);
		CUDA_TRY( hipMemcpy(h_recStart, raw_csv->recStart, recTotalSize, hipMemcpyDeviceToHost) );

		const char *h_data = (const char *)(map_data);
		unsigned long long recCount = raw_csv->num_records;

		bool quotation = false;
		for (size_t i = 1; i < raw_csv->num_records; ++i) {
			if (h_data[h_recStart[i] - 1] == raw_csv->quotechar) {
				quotation = !quotation;
				h_recStart[i] = raw_csv->num_bytes;
				recCount--;
			}
			else if (quotation) {
				h_recStart[i] = raw_csv->num_bytes;
				recCount--;
			}
		}

		CUDA_TRY( hipMemcpy(raw_csv->recStart, h_recStart, recTotalSize, hipMemcpyHostToDevice) );
		thrust::sort(thrust::device, raw_csv->recStart, raw_csv->recStart + raw_csv->num_records + 1);
		raw_csv->num_records = recCount;
		
		free(h_recStart);
	}

	//-----------------------------------------------------------------------------
	//-- Acquire header row of 

	int h_num_cols=0, h_dup_cols_removed=0;

	int skip_header=0;

	// Check if the user gave us a list of column names
	if(args->names==NULL){

		// Getting the first row of data from the file. We will parse the data to find lineterminator as
		// well as the column delimiter.
		char* cmap_data = (char *)map_data;

		unsigned long long c=0;

		raw_csv->header_row=0;
		if (args->header>=0){
			raw_csv->header_row = args->header;
		}
		if(raw_csv->header_row > (long)raw_csv->num_records){
			checkError(GDF_FILE_ERROR, "Number of records is smaller than the id of the specified header row");
		}

		unsigned long long headerPositions[2];
		CUDA_TRY( hipMemcpy(headerPositions,raw_csv->recStart + raw_csv->header_row, sizeof(unsigned long long)*2, hipMemcpyDeviceToHost));
		unsigned long long start = headerPositions[0];
		unsigned long long stop  = headerPositions[1];

		c=start;
		while(c<stop){
			if (cmap_data[c]==args->lineterminator){
				h_num_cols++;
				break;
			}
			else if(cmap_data[c] == '\r' && (c+1L)<(unsigned long long)raw_csv->num_bytes && cmap_data[c+1] == '\n'){
				h_num_cols++;
				break;
			}else if (cmap_data[c]==args->delimiter)
				h_num_cols++;
			c++;
		}

		unsigned long long prev=0;
		c=start;

		raw_csv->col_names.clear();

		if(args->header>=0){
			h_num_cols=0;
			// Storing the names of the columns into a vector of strings
			while(c<=stop){
				if (cmap_data[c]==args->delimiter || cmap_data[c]==args->lineterminator){
					std::string colName(cmap_data +prev,c-prev );
					prev=c+1;
					raw_csv->col_names.push_back(colName);
					h_num_cols++;
				}
				c++;
			}
			skip_header=1;
		}else{
			for (int i = 0; i<h_num_cols; i++){
				std::string newColName = std::to_string(i);
				raw_csv->col_names.push_back(newColName);
			}
		}
		// Allocating a boolean array that will use to state if a column needs to read or filtered.


		raw_csv->h_parseCol = (bool*)malloc(sizeof(bool) * (h_num_cols));
		RMM_TRY( RMM_ALLOC((void**)&raw_csv->d_parseCol,(sizeof(bool) * (h_num_cols)),0 ) );
		for (int i = 0; i<h_num_cols; i++)
			raw_csv->h_parseCol[i]=true;

		// Looking for duplicates
		for (auto it = raw_csv->col_names.begin(); it != raw_csv->col_names.end(); it++){
			bool found_dupe = false;
			for (auto it2 = (it+1); it2 != raw_csv->col_names.end(); it2++){
				if (*it==*it2){
					found_dupe=true;
					break;
				}
			}
			if(found_dupe){
				int count=1;
				for (auto it2 = (it+1); it2 != raw_csv->col_names.end(); it2++){
					if (*it==*it2){
						if(args->mangle_dupe_cols){
							// Replace all the duplicates of column X with X.1,X.2,... First appearance stays as X.
							std::string newColName  = *it2;
							newColName += "." + std::to_string(count); 
							count++;
							*it2 = newColName;							
						} else{
							// All duplicate fields will be ignored.
							int pos=std::distance(raw_csv->col_names.begin(), it2);
							raw_csv->h_parseCol[pos]=false;
							h_dup_cols_removed++;
						}
					}
				}
			}
		}

		raw_csv->num_actual_cols = h_num_cols;							// Actuaul number of columns in the CSV file
		raw_csv->num_active_cols = h_num_cols-h_dup_cols_removed;		// Number of fields that need to be processed based on duplicatation fields

		CUDA_TRY(hipMemcpy(raw_csv->d_parseCol, raw_csv->h_parseCol, sizeof(bool) * (h_num_cols), hipMemcpyHostToDevice));
	}
	else {
		raw_csv->h_parseCol = (bool*)malloc(sizeof(bool) * (args->num_cols));
		RMM_TRY( RMM_ALLOC((void**)&raw_csv->d_parseCol,(sizeof(bool) * (args->num_cols)),0 ) );

		for (int i = 0; i<raw_csv->num_actual_cols; i++){
			raw_csv->h_parseCol[i]=true;
			std::string col_name 	= args->names[i];
			raw_csv->col_names.push_back(col_name);

		}
		CUDA_TRY(hipMemcpy(raw_csv->d_parseCol, raw_csv->h_parseCol, sizeof(bool) * (args->num_cols), hipMemcpyHostToDevice));
	}

	// User can give
	if (args->use_cols_int!=NULL || args->use_cols_char!=NULL){
		if(args->use_cols_int!=NULL){
			for (int i = 0; i<raw_csv->num_actual_cols; i++)
				raw_csv->h_parseCol[i]=false;
			for(int i=0; i < args->use_cols_int_len; i++){
				int pos = args->use_cols_int[i];
				raw_csv->h_parseCol[pos]=true;
			}
			raw_csv->num_active_cols = args->use_cols_int_len;
		}else{
			for (int i = 0; i<raw_csv->num_actual_cols; i++)
				raw_csv->h_parseCol[i]=false;
			int countFound=0;
			for(int i=0; i < args->use_cols_char_len; i++){
				std::string colName(args->use_cols_char[i]);
				for (auto it = raw_csv->col_names.begin(); it != raw_csv->col_names.end(); it++){
					if(colName==*it){
						countFound++;
						int pos=std::distance(raw_csv->col_names.begin(), it);
						raw_csv->h_parseCol[pos]=true;
						break;
					}
				}
			}
			raw_csv->num_active_cols = countFound;
		}
		CUDA_TRY(hipMemcpy(raw_csv->d_parseCol, raw_csv->h_parseCol, sizeof(bool) * (raw_csv->num_actual_cols), hipMemcpyHostToDevice));
	}

	raw_csv->num_records -= (args->skiprows + args->skipfooter); 
	if(skip_header==0){
		raw_csv->header_row=-1;
	}else{
		raw_csv->num_records-=1;
	}

	//-----------------------------------------------------------------------------
	//---  done with host data
	close(fd);
	munmap(map_data, raw_csv->num_bytes);


	//-----------------------------------------------------------------------------
	//--- Auto detect types of the vectors

	// if(args->dtype==NULL){
	if(args->names==NULL){

		column_data_t *d_ColumnData,*h_ColumnData;

		h_ColumnData = (column_data_t*)malloc(sizeof(column_data_t) * (raw_csv->num_active_cols));
		RMM_TRY( RMM_ALLOC((void**)&d_ColumnData,(sizeof(column_data_t) * (raw_csv->num_active_cols)),0 ) );

		CUDA_TRY( hipMemset(d_ColumnData,	0, 	(sizeof(column_data_t) * (raw_csv->num_active_cols)) ) ) ;

		launch_dataTypeDetection(raw_csv, args->skiprows, d_ColumnData);

		CUDA_TRY( hipMemcpy(h_ColumnData,d_ColumnData, sizeof(column_data_t) * (raw_csv->num_active_cols), hipMemcpyDeviceToHost));

	    vector<gdf_dtype>	d_detectedTypes;			// host: array of dtypes (since gdf_columns are not created until end)

		raw_csv->dtypes.clear();

		for(int col = 0; col < raw_csv->num_active_cols; col++){
			unsigned long long countInt = h_ColumnData[col].countInt8+h_ColumnData[col].countInt16+
										  h_ColumnData[col].countInt32+h_ColumnData[col].countInt64;

			if (h_ColumnData[col].countNULL == raw_csv->num_records){
				d_detectedTypes.push_back(GDF_INT8); // Entire column is NULL. Allocating the smallest amount of memory
			} else if(h_ColumnData[col].countString>0L){
				d_detectedTypes.push_back(GDF_CATEGORY); // For auto-detection, we are currently not supporting strings.
			} else if(h_ColumnData[col].countDateAndTime>0L){
				d_detectedTypes.push_back(GDF_DATE64);
			} else if(h_ColumnData[col].countFloat > 0L  ||  
				(h_ColumnData[col].countFloat==0L && countInt >0L && h_ColumnData[col].countNULL >0L) ) {
				// The second condition has been added to conform to PANDAS which states that a colum of 
				// integers with a single NULL record need to be treated as floats.
				d_detectedTypes.push_back(GDF_FLOAT64);
			}
			else { 
				d_detectedTypes.push_back(GDF_INT64);
			}
		}

		raw_csv->dtypes=d_detectedTypes;

		free(h_ColumnData);
		RMM_TRY( RMM_FREE( d_ColumnData, 0 ) );
	}
	else{
		for ( int x = 0; x < raw_csv->num_actual_cols; x++) {

			std::string temp_type 	= args->dtype[x];
			gdf_dtype col_dtype		= convertStringToDtype( temp_type );

			if (col_dtype == GDF_invalid)
				return GDF_UNSUPPORTED_DTYPE;

			raw_csv->dtypes.push_back(col_dtype);
		}
	}


	//-----------------------------------------------------------------------------
	//--- allocate space for the results
	gdf_column **cols = (gdf_column **)malloc( sizeof(gdf_column *) * raw_csv->num_active_cols);

	void **d_data,**h_data;
	gdf_valid_type **d_valid,**h_valid;
    unsigned long long	*d_valid_count,*h_valid_count;
	gdf_dtype *d_dtypes,*h_dtypes;





	h_dtypes 		= (gdf_dtype*)malloc (	sizeof(gdf_dtype)* (raw_csv->num_active_cols));
	h_valid_count	= (unsigned long long*)malloc (	sizeof(unsigned long long)* (raw_csv->num_active_cols));
	h_data 			= (void**)malloc (	sizeof(void*)* (raw_csv->num_active_cols));
	h_valid 		= (gdf_valid_type**)malloc (	sizeof(gdf_valid_type*)* (raw_csv->num_active_cols));

	RMM_TRY( RMM_ALLOC((void**)&d_dtypes, 		(sizeof(gdf_dtype) 			* raw_csv->num_active_cols), 0 ) );
	RMM_TRY( RMM_ALLOC((void**)&d_data, 		(sizeof(void *)				* raw_csv->num_active_cols), 0 ) );
	RMM_TRY( RMM_ALLOC((void**)&d_valid, 		(sizeof(gdf_valid_type *)	* raw_csv->num_active_cols), 0 ) );
	RMM_TRY( RMM_ALLOC((void**)&d_valid_count, 	(sizeof(unsigned long long) * raw_csv->num_active_cols), 0 ) );
	CUDA_TRY( hipMemset(d_valid_count,	0, 		(sizeof(unsigned long long)	* raw_csv->num_active_cols)) );


	int stringColCount=0;
	for (int col = 0; col < raw_csv->num_active_cols; col++) {
		if(raw_csv->dtypes[col]==gdf_dtype::GDF_STRING)
			stringColCount++;
	}

	string_pair **h_str_cols = NULL, **d_str_cols = NULL;

	if (stringColCount > 0 ) {
		h_str_cols = (string_pair**) malloc ((sizeof(string_pair *)	* stringColCount));
		RMM_TRY( RMM_ALLOC((void**)&d_str_cols, 	(sizeof(string_pair *)		* stringColCount), 0) );

		for (int col = 0; col < stringColCount; col++) {
			RMM_TRY( RMM_ALLOC((void**)(h_str_cols + col), sizeof(string_pair) * (raw_csv->num_records), 0) );
		}

		CUDA_TRY(hipMemcpy(d_str_cols, h_str_cols, sizeof(string_pair *)	* stringColCount, hipMemcpyHostToDevice));
	}

	for (int col = 0; col < raw_csv->num_active_cols; col++) {

		gdf_column *gdf = (gdf_column *)malloc(sizeof(gdf_column) * 1);

		gdf->size		= raw_csv->num_records;
		gdf->dtype		= raw_csv->dtypes[col];
		gdf->null_count	= 0;						// will be filled in later

		//--- column name
		std::string str = raw_csv->col_names[col];
		int len = str.length() + 1;
		gdf->col_name = (char *)malloc(sizeof(char) * len);
		memcpy(gdf->col_name, str.c_str(), len);
		gdf->col_name[len -1] = '\0';

		allocateGdfDataSpace(gdf);

		cols[col] 		= gdf;
		h_dtypes[col] 	= raw_csv->dtypes[col];
		h_data[col] 	= gdf->data;
		h_valid[col] 	= gdf->valid;
	}
	CUDA_TRY( hipMemcpy(d_dtypes,h_dtypes, sizeof(gdf_dtype) * (raw_csv->num_active_cols), hipMemcpyHostToDevice));
	CUDA_TRY( hipMemcpy(d_data,h_data, sizeof(void*) * (raw_csv->num_active_cols), hipMemcpyHostToDevice));
	CUDA_TRY( hipMemcpy(d_valid,h_valid, sizeof(gdf_valid_type*) * (raw_csv->num_active_cols), hipMemcpyHostToDevice));

	free(h_dtypes); 
	free(h_valid); 
	free(h_data); 
	
	launch_dataConvertColumns(raw_csv,d_data, d_valid, d_dtypes,d_str_cols, args->skiprows, d_valid_count);
	hipDeviceSynchronize();

	stringColCount=0;
	for (int col = 0; col < raw_csv->num_active_cols; col++) {

		gdf_column *gdf = cols[col];

		if (gdf->dtype != gdf_dtype::GDF_STRING)
			continue;

		NVStrings* const stringCol = NVStrings::create_from_index(h_str_cols[stringColCount],size_t(raw_csv->num_records));
		if ((raw_csv->quotechar != '\0') && (raw_csv->doublequote==true)) {
			// In PANDAS, default of enabling doublequote for two consecutive
			// quotechar in quote fields results in reduction to single
			std::string quotechar = std::string(&raw_csv->quotechar);
			std::string doublequotechar = quotechar + raw_csv->quotechar;
			gdf->data = stringCol->replace(doublequotechar.c_str(), quotechar.c_str());
			NVStrings::destroy(stringCol);
		}
		else {
			gdf->data = stringCol;
		}

		RMM_TRY( RMM_FREE( h_str_cols [stringColCount], 0 ) );

		stringColCount++;
	}


	CUDA_TRY( hipMemcpy(h_valid_count,d_valid_count, sizeof(unsigned long long) * (raw_csv->num_active_cols), hipMemcpyDeviceToHost));

	//--- set the null count
	for ( int col = 0; col < raw_csv->num_active_cols; col++) {
		cols[col]->null_count = raw_csv->num_records - h_valid_count[col];
	}

	free(h_valid_count); 

	// free up space that is no longer needed
	if (h_str_cols != NULL)
		free ( h_str_cols);

	free(raw_csv->h_parseCol);

	if (d_str_cols != NULL)
		RMM_TRY( RMM_FREE( d_str_cols, 0 ) ); 

	RMM_TRY( RMM_FREE( d_valid, 0 ) );
	RMM_TRY( RMM_FREE( d_valid_count, 0 ) );
	RMM_TRY( RMM_FREE( d_dtypes, 0 ) );
	RMM_TRY( RMM_FREE( d_data, 0 ) ); 

	RMM_TRY( RMM_FREE( raw_csv->recStart, 0 ) ); 
	RMM_TRY( RMM_FREE( raw_csv->d_parseCol, 0 ) ); 
	RMM_TRY( RMM_FREE( raw_csv->d_num_records, 0 ) ); 
	CUDA_TRY( hipFree ( raw_csv->data) );


	args->data 			= cols;
	args->num_cols_out	= raw_csv->num_active_cols;
	args->num_rows_out	= raw_csv->num_records;

	delete raw_csv;
	return error;
}



/*
 * What is passed in is the data type as a string, need to convert that into gdf_dtype enum
 */
gdf_dtype convertStringToDtype(std::string &dtype) {

	if (dtype.compare( "str") == 0) 		return GDF_STRING;
	if (dtype.compare( "date") == 0) 		return GDF_DATE64;
	if (dtype.compare( "date32") == 0) 		return GDF_DATE32;
	if (dtype.compare( "date64") == 0) 		return GDF_DATE64;
	if (dtype.compare( "timestamp") == 0)	return GDF_TIMESTAMP;
	if (dtype.compare( "category") == 0) 	return GDF_CATEGORY;
	if (dtype.compare( "float") == 0)		return GDF_FLOAT32;
	if (dtype.compare( "float32") == 0)		return GDF_FLOAT32;
	if (dtype.compare( "float64") == 0)		return GDF_FLOAT64;
	if (dtype.compare( "double") == 0)		return GDF_FLOAT64;
	if (dtype.compare( "short") == 0)		return GDF_INT16;
	if (dtype.compare( "int") == 0)			return GDF_INT32;
	if (dtype.compare( "int32") == 0)		return GDF_INT32;
	if (dtype.compare( "int64") == 0)		return GDF_INT64;
	if (dtype.compare( "long") == 0)		return GDF_INT64;

	return GDF_invalid;
}


/*
 * Create the raw_csv_t structure and allocate space on the GPU
 */
gdf_error updateRawCsv( const char * data, long num_bytes, raw_csv_t * raw ) {

	int num_bits = (num_bytes + 63) / 64;

	CUDA_TRY( hipMallocManaged ((void**)&raw->data, 		(sizeof(char)		* num_bytes)));
	// RMM_TRY( RMM_ALLOC((void**)&raw->data, 		(sizeof(char)		* num_bytes),0 ));

	RMM_TRY( RMM_ALLOC((void**)&raw->d_num_records, sizeof(unsigned long long),0) );

	CUDA_TRY( hipMemcpy(raw->data, data, num_bytes, hipMemcpyHostToDevice));
	CUDA_TRY( hipMemset(raw->d_num_records,0, ((sizeof(long)) )) );

	raw->num_bits  = num_bits;

	return GDF_SUCCESS;
}


/*
 * For each of the gdf_cvolumns, create the on-device space.  the on-host fields should already be filled in
 */
gdf_error allocateGdfDataSpace(gdf_column *gdf) {

	long N = gdf->size;
	long num_bitmaps = (N + 31) / 8;			// 8 bytes per bitmap

	//--- allocate space for the valid bitmaps
	RMM_TRY( RMM_ALLOC((void**)&gdf->valid, (sizeof(gdf_valid_type) * num_bitmaps), 0) );
	CUDA_TRY(hipMemset(gdf->valid, 0, (sizeof(gdf_valid_type) 	* num_bitmaps)) );

	int elementSize=0;
	//--- Allocate space for the data
	switch(gdf->dtype) {
		case gdf_dtype::GDF_INT8:
			elementSize = sizeof(int8_t);
			break;
		case gdf_dtype::GDF_INT16:
			elementSize = sizeof(int16_t);
			break;
		case gdf_dtype::GDF_INT32:
			elementSize = sizeof(int32_t);
			break;
		case gdf_dtype::GDF_INT64:
			elementSize = sizeof(int64_t);
			break;
		case gdf_dtype::GDF_FLOAT32:
			elementSize = sizeof(float);
			break;
		case gdf_dtype::GDF_FLOAT64:
			elementSize = sizeof(double);
			break;
		case gdf_dtype::GDF_DATE32:
			elementSize = sizeof(gdf_date32);
			break;
		case gdf_dtype::GDF_DATE64:
			elementSize = sizeof(gdf_date64);
			break;
		case gdf_dtype::GDF_TIMESTAMP:
			elementSize = sizeof(int64_t);
			break;
		case gdf_dtype::GDF_CATEGORY:
			elementSize = sizeof(gdf_category);
			break;
		case gdf_dtype::GDF_STRING:
			return gdf_error::GDF_SUCCESS;
			// Memory for gdf->data allocated by string class eventually
		default:
			return GDF_UNSUPPORTED_DTYPE;
	}
	
	RMM_TRY( RMM_ALLOC((void**)&gdf->data, elementSize * N, 0) );

	return gdf_error::GDF_SUCCESS;
}


//----------------------------------------------------------------------------------------------------------------
//				CUDA Kernels
//----------------------------------------------------------------------------------------------------------------


gdf_error launch_countRecords(raw_csv_t * csvData) {

	int blockSize;		// suggested thread count to use
	int minGridSize;	// minimum block count required
	CUDA_TRY( hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, countRecords) );

	// Calculate actual block count to use based on bitmap count
	// Each bitmap is for a 64-byte chunk, and each data index is bitmap ID * 64
	int gridSize = (csvData->num_bits + blockSize - 1) / blockSize;

	countRecords <<< gridSize, blockSize >>> (
		csvData->data, csvData->terminator, csvData->quotechar,
		csvData->num_bytes, csvData->num_bits, csvData->d_num_records
	);

	CUDA_TRY(hipGetLastError());

	long recs=-1;
	CUDA_TRY(hipMemcpy(&recs, csvData->d_num_records, sizeof(long), hipMemcpyDeviceToHost));
	csvData->num_records=recs;

	CUDA_TRY(hipGetLastError());

	return GDF_SUCCESS;
}


__global__ void countRecords(char *data, const char terminator, const char quotechar, long num_bytes, long num_bits, unsigned long long* num_records) {

	// thread IDs range per block, so also need the block id
	long tid = threadIdx.x + (blockDim.x * blockIdx.x);

	if (tid >= num_bits)
		return;

	// data ID is a multiple of 64
	long did = tid * 64L;

	char *raw = (data + did);

	long byteToProcess = ((did + 64L) < num_bytes) ? 64L : (num_bytes - did);

	// process the data
	long tokenCount = 0;
	for (long x = 0; x < byteToProcess; x++) {
		
		// Scan and log records. If quotations are enabled, then also log quotes
		// for a postprocess ignore, as the chunk here has limited visibility.
		if ((raw[x] == terminator) || (quotechar != '\0' && raw[x] == quotechar)) {
			tokenCount++;
		} else if (raw[x] == '\r' && (x+1L)<num_bytes && raw[x +1] == '\n') {
			x++;
			tokenCount++;
		}

	}
	atomicAdd((unsigned long long int*)num_records,(unsigned long long int)tokenCount);
}


gdf_error launch_storeRecordStart(raw_csv_t * csvData) {

	int blockSize;		// suggested thread count to use
	int minGridSize;	// minimum block count required
	CUDA_TRY( hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, storeRecordStart) );

	// Calculate actual block count to use based on bitmap count
	// Each bitmap is for a 64-byte chunk, and each data index is bitmap ID * 64
	int gridSize = (csvData->num_bits + blockSize - 1) / blockSize;

	storeRecordStart <<< gridSize, blockSize >>> (
		csvData->data, csvData->terminator, csvData->quotechar,
		csvData->num_bytes, csvData->num_bits, csvData->d_num_records,
		csvData->recStart
	);

	CUDA_TRY( hipGetLastError() );
	return GDF_SUCCESS;
}


__global__ void storeRecordStart(char *data, const char terminator, const char quotechar, long num_bytes, long num_bits, unsigned long long* num_records,unsigned long long* recStart) {

	// thread IDs range per block, so also need the block id
	long tid = threadIdx.x + (blockDim.x * blockIdx.x);

	if ( tid >= num_bits)
		return;

	// data ID - multiple of 64
	long did = tid * 64L;

	char *raw = (data + did);

	long byteToProcess = ((did + 64L) < num_bytes) ? 64L : (num_bytes - did);

	if(tid==0){
		long pos = atomicAdd((unsigned long long int*)num_records,(unsigned long long int)1);
		recStart[pos]=did+0;
	}

	// process the data
	for (long x = 0; x < byteToProcess; x++) {

		// Scan and log records. If quotations are enabled, then also log quotes
		// for a postprocess ignore, as the chunk here has limited visibility.
		if ((raw[x] == terminator) || (quotechar != '\0' && raw[x] == quotechar)) {

			long pos = atomicAdd((unsigned long long int*)num_records,(unsigned long long int)1);
			recStart[pos]=did+x+1;

		} else if (raw[x] == '\r' && (x+1L)<num_bytes && raw[x +1] == '\n') {

			x++;
			long pos = atomicAdd((unsigned long long int*)num_records,(unsigned long long int)1);
			recStart[pos]=did+x+1;
		}

	}
}


//----------------------------------------------------------------------------------------------------------------


gdf_error launch_dataConvertColumns(raw_csv_t *raw_csv, void **gdf, gdf_valid_type** valid, gdf_dtype* d_dtypes,string_pair **str_cols, long row_offset, unsigned long long *num_valid) {

	int blockSize;		// suggested thread count to use
	int minGridSize;	// minimum block count required
	CUDA_TRY( hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, convertCsvToGdf) );

	// Calculate actual block count to use based on records count
	int gridSize = (raw_csv->num_records + blockSize - 1) / blockSize;

	parsing_opts_t opts;
	opts.delimiter		= raw_csv->delimiter;
	opts.terminator		= raw_csv->terminator;
	opts.quotechar		= raw_csv->quotechar;
	opts.keepquotes		= raw_csv->keepquotes;
	opts.decimal		= raw_csv->decimal;
	opts.thousands		= raw_csv->thousands;

	convertCsvToGdf <<< gridSize, blockSize >>>(
		raw_csv->data,
		opts,
		raw_csv->num_records,
		raw_csv->num_actual_cols,
		raw_csv->d_parseCol,
		raw_csv->recStart,
		d_dtypes,
		gdf,
		valid,
		str_cols,
		row_offset,
		raw_csv->header_row,
		raw_csv->dayfirst,
		num_valid
	);

	CUDA_TRY( hipGetLastError() );
	return GDF_SUCCESS;
}


/*
 * Data is processed in one row\record at a time - so the number of total threads (tid) is equal to the number of rows.
 *
 */
__global__ void convertCsvToGdf(
		char 			*raw_csv,
		const parsing_opts_t	 	opts,
		unsigned long long  num_records,
		int  			num_columns,
		bool  			*parseCol,
		unsigned long long 			*recStart,
		gdf_dtype 		*dtype,
		void			**gdf_data,
		gdf_valid_type 	**valid,
		string_pair		**str_cols,
		unsigned long long 			row_offset,
		long 			header_row,
		bool			dayfirst,
		unsigned long long			*num_valid
		)
{
	// thread IDs range per block, so also need the block id
	long	rec_id  = threadIdx.x + (blockDim.x * blockIdx.x);		// this is entry into the field array - tid is an elements within the num_entries array

	// we can have more threads than data, make sure we are not past the end of the data
	if ( rec_id >= num_records)
		return;

	long extraOff=0;
	if(rec_id>=header_row && header_row>=0)
		extraOff=1;

	long start 		= recStart[rec_id + row_offset + extraOff];
	long stop 		= recStart[rec_id + 1 + row_offset + extraOff];

	long pos 		= start;
	int  col 		= 0;
	int  actual_col = 0;
	int  stringCol 	= 0;
	bool quotation	= false;

	while(col<num_columns){

		if(start>stop)
			break;

		while(true){
			// Use simple logic to ignore control chars between any quote seq
			// Handles nominal cases including doublequotes within quotes, but
			// may not output exact failures as PANDAS for malformed fields
			if(raw_csv[pos] == opts.quotechar){
				quotation = !quotation;
			}
			else if(quotation==false){
				if(raw_csv[pos] == opts.delimiter){
					break;
				}
				else if(raw_csv[pos] == opts.terminator){
					break;
				}
				else if(raw_csv[pos] == '\r' && ((pos+1) < stop && raw_csv[pos+1] == '\n')){
					stop--;
					break;
				}
			}
			if(pos>=stop)
				break;
			pos++;
		}

		if(parseCol[col]==true){

			long tempPos=pos-1;

			if(dtype[col] != gdf_dtype::GDF_CATEGORY && dtype[col] != gdf_dtype::GDF_STRING){
				removePrePostWhiteSpaces2(raw_csv, &start, &tempPos);
			}


			if(start<=(tempPos)) { // Empty strings are not legal values

				switch(dtype[col]) {
					case gdf_dtype::GDF_INT8:
					{
						int8_t *gdf_out = (int8_t *)gdf_data[actual_col];
						gdf_out[rec_id] = convertStrtoInt<int8_t>(raw_csv, start, tempPos, opts.thousands);
					}
						break;
					case gdf_dtype::GDF_INT16: {
						int16_t *gdf_out = (int16_t *)gdf_data[actual_col];
						gdf_out[rec_id] = convertStrtoInt<int16_t>(raw_csv, start, tempPos, opts.thousands);
					}
						break;
					case gdf_dtype::GDF_INT32:
					{
						int32_t *gdf_out = (int32_t *)gdf_data[actual_col];
						gdf_out[rec_id] = convertStrtoInt<int32_t>(raw_csv, start, tempPos, opts.thousands);
					}
						break;
					case gdf_dtype::GDF_INT64:
					{
						int64_t *gdf_out = (int64_t *)gdf_data[actual_col];
						gdf_out[rec_id] = convertStrtoInt<int64_t>(raw_csv, start, tempPos, opts.thousands);
					}
						break;
					case gdf_dtype::GDF_FLOAT32:
					{
						float *gdf_out = (float *)gdf_data[actual_col];
						gdf_out[rec_id] = convertStrtoFloat<float>(raw_csv, start, tempPos, opts.decimal, opts.thousands);
					}
						break;
					case gdf_dtype::GDF_FLOAT64:
					{
						double *gdf_out = (double *)gdf_data[actual_col];
						gdf_out[rec_id] = convertStrtoFloat<double>(raw_csv, start, tempPos, opts.decimal, opts.thousands);
					}
						break;
					case gdf_dtype::GDF_DATE32:
					{
						gdf_date32 *gdf_out = (gdf_date32 *)gdf_data[actual_col];
						gdf_out[rec_id] = parseDateFormat(raw_csv, start, tempPos, dayfirst);
					}
						break;
					case gdf_dtype::GDF_DATE64:
					{
						gdf_date64 *gdf_out = (gdf_date64 *)gdf_data[actual_col];
						gdf_out[rec_id] = parseDateTimeFormat(raw_csv, start, tempPos, dayfirst);
					}
						break;
					case gdf_dtype::GDF_TIMESTAMP:
					{
						int64_t *gdf_out = (int64_t *)gdf_data[actual_col];
						gdf_out[rec_id] = convertStrtoInt<int64_t>(raw_csv, start, tempPos, opts.thousands);
					}
					break;
					case gdf_dtype::GDF_CATEGORY:
					{
						gdf_category *gdf_out = (gdf_category *)gdf_data[actual_col];
						gdf_out[rec_id] = convertStrtoHash(raw_csv, start, pos, HASH_SEED);
					}
						break;
					case gdf_dtype::GDF_STRING:
					{
						long end = pos;
						if(opts.keepquotes==false){
							if((raw_csv[start] == opts.quotechar) && (raw_csv[end-1] == opts.quotechar)){
								start++;
								end--;
							}
						}
						str_cols[stringCol][rec_id].first	= raw_csv+start;
						str_cols[stringCol][rec_id].second	= size_t(end-start);
						stringCol++;
					}
						break;
					default:
						break;
				}

				// set the valid bitmap - all bits were set to 0 to start
				int bitmapIdx 	= whichBitmap(rec_id);  	// which bitmap
				int bitIdx		= whichBit(rec_id);		// which bit - over an 8-bit index
				setBit(valid[col]+bitmapIdx, bitIdx);		// This is done with atomics

				atomicAdd((unsigned long long int*)&num_valid[col],(unsigned long long int)1);
			}
			else if(dtype[col]==gdf_dtype::GDF_STRING){
				str_cols[stringCol][rec_id].first 	= NULL;
				str_cols[stringCol][rec_id].second 	= 0;
				stringCol++;
			}
			actual_col++;
		}
		pos++;
		start=pos;
		col++;

	}
}



//----------------------------------------------------------------------------------------------------------------


gdf_error launch_dataTypeDetection(
	raw_csv_t * raw_csv, 
	long row_offset,
	column_data_t* d_columnData) 
{
	int blockSize;		// suggested thread count to use
	int minGridSize;	// minimum block count required
	CUDA_TRY( hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, dataTypeDetection) );

	// Calculate actual block count to use based on records count
	int gridSize = (raw_csv->num_records + blockSize - 1) / blockSize;

	parsing_opts_t opts;
	opts.delimiter		= raw_csv->delimiter;
	opts.terminator		= raw_csv->terminator;
	opts.quotechar		= raw_csv->quotechar;
	opts.keepquotes		= raw_csv->keepquotes;

	dataTypeDetection <<< gridSize, blockSize >>>(
		raw_csv->data,
		opts,
		raw_csv->num_records,
		raw_csv->num_actual_cols,
		raw_csv->d_parseCol,
		raw_csv->recStart,
		row_offset,
		raw_csv->header_row,
		d_columnData
	);

	CUDA_TRY( hipGetLastError() );
	return GDF_SUCCESS;
}

/*
 */
__global__ void dataTypeDetection(
		char 			*raw_csv,
		const parsing_opts_t			opts,
		unsigned long long  			num_records,
		int  			num_columns,
		bool  			*parseCol,
		unsigned long long 			*recStart,
		unsigned long long  			row_offset,
		long 			header_row,
		column_data_t* d_columnData
		)
{

	// thread IDs range per block, so also need the block id
	long	rec_id  = threadIdx.x + (blockDim.x * blockIdx.x);		// this is entry into the field array - tid is an elements within the num_entries array

	// we can have more threads than data, make sure we are not past the end of the data
	if ( rec_id >= num_records)
		return;

	long extraOff=0;
	if(rec_id>=header_row && header_row>=0)
		extraOff=1;

	long start 		= recStart[rec_id + row_offset + extraOff];
	long stop 		= recStart[rec_id + 1 + row_offset + extraOff];

	long pos 		= start;
	int  col 		= 0;
	int  actual_col = 0;
	bool quotation	= false;

	// Going through all the columns of a given record
	while(col<num_columns){

		if(start>stop)
			break;

		// Finding the breaking point for each column
		while(true){
			// Use simple logic to ignore control chars between any quote seq
			// Handles nominal cases including doublequotes within quotes, but
			// may not output exact failures as PANDAS for malformed fields
			if(raw_csv[pos] == opts.quotechar){
				quotation = !quotation;
			}
			else if(quotation==false){
				if(raw_csv[pos] == opts.delimiter){
					break;
				}
				else if(raw_csv[pos] == opts.terminator){
					break;
				}
				else if(raw_csv[pos] == '\r' && ((pos+1) < stop && raw_csv[pos+1] == '\n')){
					stop--;
					break;
				}
			}
			if(pos>=stop)
				break;
			pos++;
		}


		// Checking if this is a column that the user wants --- user can filter columns
		if(parseCol[col]==true){

			long tempPos=pos-1;

			// Checking if the record is NULL
			if(start>(tempPos)){
				atomicAdd(& d_columnData[actual_col].countNULL, 1L);
				pos++;
				start=pos;
				col++;
				actual_col++;
				continue;	
			}

			long countNumber=0;
			long countDecimal=0;
			long countSlash=0;
			long countDash=0;
			long countColon=0;
			long countString=0;

			long strLen=pos-start;

			// Remove all pre and post white-spaces.  We might find additional NULL fields if the entire entry is made up of only spaces.
			removePrePostWhiteSpaces2(raw_csv, &start, &tempPos);
			for(long startPos=start; startPos<=tempPos; startPos++){
				if(raw_csv[startPos]>= '0' && raw_csv[startPos] <= '9'){
					countNumber++;
					continue;
				}
				// Looking for unique characters that will help identify column types.
				switch (raw_csv[startPos]){
					case '.':
						countDecimal++;break;
					case '-':
						countDash++; break;
					case '/':
						countSlash++;break;
					case ':':
						countColon++;break;
					default:
						countString++;
						break;	
				}
			}

			if(strLen==0) // Removed spaces ' ' in the pre-processing and thus we can have an empty string.
				atomicAdd(& d_columnData[actual_col].countNULL, 1L);
			// Integers have to have the length of the string or can be off by one if they start with a minus sign
			else if(countNumber==(strLen) || ( strLen>1 && countNumber==(strLen-1) && raw_csv[start]=='-') ){
				// Checking to see if we the integer value requires 8,16,32,64 bits.
				// This will allow us to allocate the exact amount of memory.
				int64_t i = convertStrtoInt<int64_t>(raw_csv, start, tempPos, opts.thousands);
				if(i >= (1L<<31)){
					atomicAdd(& d_columnData[actual_col].countInt64, 1L);
				}
				else if(i >= (1L<<15)){
					atomicAdd(& d_columnData[actual_col].countInt32, 1L);
				}
				else if(i >= (1L<<7)){
					atomicAdd(& d_columnData[actual_col].countInt16, 1L);
				}
				else
					atomicAdd(& d_columnData[actual_col].countInt8, 1L);
			}
			// Floating point numbers are made up of numerical strings, have to have a decimal sign, and can have a minus sign.
			else if((countNumber==(strLen-1) && countDecimal==1) || (strLen>2 && countNumber==(strLen-2) && raw_csv[start]=='-')){
					atomicAdd(& d_columnData[actual_col].countFloat, 1L);
			}
			// The date-time field cannot have more than 3 strings. As such if an entry has more than 3 string characters, it is not 
			// a data-time field. Also, if a string has multiple decimals, then is not a legit number.
			else if(countString > 3 || countDecimal > 1){
				atomicAdd(& d_columnData[actual_col].countString, 1L);
			}
			else {
				// A date field can have either one or two '-' or '\'. A legal combination will only have one of them.
				// To simplify the process of auto column detection, we are not covering all the date-time formation permutations.
				if((countDash>0 && countDash<=2 && countSlash==0)|| (countDash==0 && countSlash>0 && 	countSlash<=2) ){
					if((countColon<=2)){
						atomicAdd(& d_columnData[actual_col].countDateAndTime, 1L);
					}
					else{
						atomicAdd(& d_columnData[actual_col].countString, 1L);					
					}
				}
				// Default field is string type.
				else{
					atomicAdd(& d_columnData[actual_col].countString, 1L);					
				}
			}
			actual_col++;
		}
		pos++;
		start=pos;
		col++;	

	}
}

//----------------------------------------------------------------------------------------------------------------

/*
 * Return which bit is set
 * x is the occurrence: 1 = first, 2 = seconds, ...
 */
__device__ int findSetBit(int tid, long num_bits, uint64_t *r_bits, int x) {

	int idx = tid;

	if ( x == 0 )
		return -1;

	int withinBitCount = 0;
	int offset = 0;
	int found  = 0;

	uint64_t bitmap = r_bits[idx];

	while (found != x)
	{
		if(bitmap == 0)
		{
			idx++;
			if (idx >= num_bits)
				return -1;
			bitmap = r_bits[idx];
			offset += 64;
			withinBitCount = 0;
		}

		if ( bitmap & 1 ) {
			found++;			//found a set bit
		}

		bitmap >>= 1;
		++withinBitCount;
	 }

	offset += withinBitCount -1;


	return offset;
}


