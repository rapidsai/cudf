#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2018, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
	 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

/**
 * @file csv-reader.cu  code to read csv data
 *
 * CSV Reader
 */


#include <hip/hip_runtime.h>

#include <algorithm>
#include <iostream>
#include <numeric>
#include <string>
#include <vector>

#include <stdio.h>
#include <stdlib.h>

#include <unistd.h>
#include <fcntl.h>
#include <sys/types.h>
#include <sys/stat.h>
#include <sys/mman.h>

#include <thrust/scan.h>
#include <thrust/reduce.h>
#include <thrust/device_ptr.h>
#include <thrust/execution_policy.h>

#include <thrust/host_vector.h>

#include "type_conversion.cuh"
#include "datetime_parser.cuh"

#include "cudf.h"
#include "utilities/error_utils.h"
#include "utilities/trie.cuh"
#include "utilities/type_dispatcher.hpp"

#include "rmm/rmm.h"
#include "rmm/thrust_rmm_allocator.h"
#include "io/comp/io_uncomp.h"

constexpr size_t max_chunk_bytes = 64*1024*1024; // 64MB

using std::vector;
using std::string;

using cu_reccnt_t = unsigned long long int;
using cu_recstart_t = unsigned long long int;


/**---------------------------------------------------------------------------*
 * @brief Struct used for internal parsing state
 *---------------------------------------------------------------------------**/
typedef struct raw_csv_ {
    char *				data;			// on-device: the raw unprocessed CSV data - loaded as a large char * array
    cu_recstart_t*		recStart;		// on-device: Starting position of the records.

    ParseOptions        opts;			// host: options to control parsing behavior

    long				num_bytes;		// host: the number of bytes in the data
    long				num_bits;		// host: the number of 64-bit bitmaps (different than valid)
	gdf_size_type 		num_records;  	// host: number of records loaded into device memory, and then number of records to read
	// int				num_cols;		// host: number of columns
	int					num_active_cols;	// host: number of columns that will be return to user.
	int					num_actual_cols;	// host: number of columns in the file --- based on the number of columns in header
    vector<gdf_dtype>	dtypes;			// host: array of dtypes (since gdf_columns are not created until end)
    vector<string>		col_names;		// host: array of column names
    bool* 				h_parseCol;		// host   : array of booleans stating if column should be parsed in reading process: parseCol[x]=false means that the column x needs to be filtered out.
    bool* 				d_parseCol;		// device : array of booleans stating if column should be parsed in reading process: parseCol[x]=false means that the column x needs to be filtered out.

    long        byte_range_offset;  // offset into the data to start parsing
    long        byte_range_size;    // length of the data of interest to parse

    gdf_size_type header_row;       ///< host: Row index of the header
    gdf_size_type nrows;            ///< host: Number of rows to read. -1 for all rows
    gdf_size_type skiprows;         ///< host: Number of rows to skip from the start
    gdf_size_type skipfooter;       ///< host: Number of rows to skip from the end
    std::vector<char> header;       ///< host: Header row data, for parsing column names
    string prefix;                  ///< host: Prepended to column ID if there is no header or input column names

    rmm::device_vector<int32_t>	d_trueValues;		// device: array of values to recognize as true
    rmm::device_vector<int32_t>	d_falseValues;		// device: array of values to recognize as false
    rmm::device_vector<SerialTrieNode>	d_naTrie;	// device: serialized trie of NA values
} raw_csv_t;

typedef struct column_data_ {
	unsigned long long countFloat;
	unsigned long long countDateAndTime;
	unsigned long long countString;
	unsigned long long countInt8;
	unsigned long long countInt16;
	unsigned long long countInt32;
	unsigned long long countInt64;
	gdf_size_type countNULL;
} column_data_t;

using string_pair = std::pair<const char*,size_t>;

//
//---------------create and process ---------------------------------------------
//
gdf_error parseArguments(csv_read_arg *args, raw_csv_t *csv);
// gdf_error getColNamesAndTypes(const char **col_names, const  char **dtypes, raw_csv_t *d);
gdf_error inferCompressionType(const char* compression_arg, const char* filepath, string& compression_type);
gdf_error getUncompressedHostData(const char* h_data, size_t num_bytes, 
	const string& compression, 
	vector<char>& h_uncomp_data);
gdf_error uploadDataToDevice(const char* h_uncomp_data, size_t h_uncomp_size, raw_csv_t * raw_csv);
gdf_error allocateGdfDataSpace(gdf_column *);
gdf_dtype convertStringToDtype(std::string &dtype);

#define checkError(error, txt)  if ( error != GDF_SUCCESS) { std::cerr << "ERROR:  " << error <<  "  in "  << txt << std::endl;  return error; }

//
//---------------CUDA Kernel ---------------------------------------------
//

__device__ int findSetBit(int tid, long num_bits, uint64_t *f_bits, int x);

gdf_error launch_countRecords(const char* h_data, size_t h_size, raw_csv_t * raw_csv, gdf_size_type& rec_cnt);
gdf_error launch_storeRecordStart(const char* h_data, size_t h_size, raw_csv_t * csvData);
gdf_error launch_dataConvertColumns(raw_csv_t * raw_csv, void** d_gdf,  gdf_valid_type** valid, gdf_dtype* d_dtypes, string_pair **str_cols, unsigned long long *);

gdf_error launch_dataTypeDetection(raw_csv_t * raw_csv, column_data_t* d_columnData);

__global__ void countRecords(char *data, const char terminator, const char quotechar, long num_bytes, long num_bits, cu_reccnt_t* num_records);
__global__ void storeRecordStart(char *data, size_t chunk_offset, 
	const char terminator, const char quotechar, bool include_first_row,
	long num_bytes, long num_bits, cu_reccnt_t* num_records,
	cu_recstart_t* recStart);
__global__ void convertCsvToGdf(char *csv, const ParseOptions opts,
	gdf_size_type num_records, int num_columns, bool *parseCol,
	cu_recstart_t *recStart, gdf_dtype *dtype, SerialTrieNode *na_trie, void **gdf_data, gdf_valid_type **valid,
	string_pair **str_cols, unsigned long long *num_valid);
__global__ void dataTypeDetection(char *raw_csv, const ParseOptions opts,
	gdf_size_type num_records, int num_columns, bool *parseCol,
	cu_recstart_t *recStart, column_data_t* d_columnData);

//
//---------------CUDA Valid (8 blocks of 8-bits) Bitmap Kernels ---------------------------------------------
//
__device__ int whichBitmap(int record) { return (record/8);  }
__device__ int whichBit(int bit) { return (bit % 8);  }

__inline__ __device__ void validAtomicOR(gdf_valid_type* address, gdf_valid_type val)
{
	int32_t *base_address = (int32_t*)((gdf_valid_type*)address - ((size_t)address & 3));
	int32_t int_val = (int32_t)val << (((size_t) address & 3) * 8);

	atomicOr(base_address, int_val);
}

__device__ void setBit(gdf_valid_type* address, int bit) {
	gdf_valid_type bitMask[8] 		= {1, 2, 4, 8, 16, 32, 64, 128};
	validAtomicOR(address, bitMask[bit]);
}


/**---------------------------------------------------------------------------*
 * @brief Estimates the maximum expected length or a row, based on the number 
 * of columns
 * 
 * If the number of columns is not available, it will return a value large 
 * enough for most use cases
 * 
 * @param[in] num_columns Number of columns in the CSV file (optional)
 * 
 * @return Estimated maximum size of a row, in bytes
 *---------------------------------------------------------------------------**/
 constexpr size_t calculateMaxRowSize(int num_columns=0) noexcept {
	constexpr size_t max_row_bytes = 16*1024; // 16KB
	constexpr size_t column_bytes = 64;
	constexpr size_t base_padding = 1024; // 1KB
	if (num_columns == 0){
		// Use flat size if the number of columns is not known
		return max_row_bytes;
	}
	else {
		// Expand the size based on the number of columns, if available
		return base_padding + num_columns * column_bytes; 
	}
}
/**
* @brief Removes the first and Last quote in the string
*/
string removeQuotes(string str, char quotechar) {
	// Exclude first and last quotation char
	const size_t first_quote = str.find(quotechar);
	if (first_quote != string::npos) {
		str.erase(first_quote, 1);
	}
	const size_t  last_quote = str.rfind(quotechar);
	if (last_quote != string::npos) {
		str.erase(last_quote, 1);
	}

	return str;
}

/**
 * @brief Parse the first row to set the column names in the raw_csv parameter 
 *
 * The first row can be either the header row, or the first data row
 *
 * @param[in,out] raw_csv Structure containing the csv parsing parameters
 * and intermediate results
 * 
 * @return gdf_error with error code on failure, otherwise GDF_SUCCESS
*/
gdf_error setColumnNamesFromCsv(raw_csv_t* raw_csv) {
	vector<char> first_row = raw_csv->header;
	// No header, read the first data row
	if (first_row.empty()) {
		cu_recstart_t first_row_len{};
		// If file only contains one row, raw_csv->recStart[1] is not valid
		if (raw_csv->num_records > 1) {
			CUDA_TRY(hipMemcpy(&first_row_len, raw_csv->recStart + 1, sizeof(cu_recstart_t), hipMemcpyDefault));
		}
		else {
			// File has one row - use the file size for the row size
			first_row_len = raw_csv->num_bytes / sizeof(char);
		}
		first_row.resize(first_row_len);
		CUDA_TRY(hipMemcpy(first_row.data(), raw_csv->data, raw_csv->num_bytes, hipMemcpyDefault));
	}

	int num_cols = 0;

	bool quotation	= false;
	for (size_t pos = 0, prev = 0; pos < first_row.size(); ++pos) {
		// Flip the quotation flag if current character is a quotechar
		if(first_row[pos] == raw_csv->opts.quotechar) {
			quotation = !quotation;
		}
		else if (!quotation &&
				 (first_row[pos] == raw_csv->opts.delimiter ||
				 first_row[pos] == raw_csv->opts.terminator)) {
			// Got to the end of a column
			if (raw_csv->header_row >= 0) {
				// first_row is the header, add the column name
				string new_col_name(first_row.data() + prev, pos - prev);
				raw_csv->col_names.push_back(removeQuotes(new_col_name, raw_csv->opts.quotechar));
			}
			else {
				// first_row is the first data row, add the automatically generated name
				raw_csv->col_names.push_back(raw_csv->prefix + std::to_string(num_cols));
			}
			num_cols++;

			// Skip adjacent delimiters if delim_whitespace is set
			while (raw_csv->opts.multi_delimiter &&
				   pos < first_row.size() &&
				   first_row[pos] == raw_csv->opts.delimiter && 
				   first_row[pos + 1] == raw_csv->opts.delimiter) {
				++pos;
			}
			prev = pos + 1;
		}
	}
	return GDF_SUCCESS;
}

/**---------------------------------------------------------------------------*
 * @brief Read in a CSV file, extract all fields and return 
 * a GDF (array of gdf_columns)
 *
 * @param[in,out] args Structure containing both the the input arguments 
 * and the returned data
 *
 * @return gdf_error
 *---------------------------------------------------------------------------**/
gdf_error read_csv(csv_read_arg *args)
{
	gdf_error error = gdf_error::GDF_SUCCESS;

	//-----------------------------------------------------------------------------
	// create the CSV data structure - this will be filled in as the CSV data is processed.
	// Done first to validate data types
	raw_csv_t * raw_csv = new raw_csv_t;
	// error = parseArguments(args, raw_csv);
	raw_csv->num_actual_cols	= args->num_cols;
	raw_csv->num_active_cols	= args->num_cols;
	raw_csv->num_records		= 0;

	raw_csv->header_row = args->header;
	raw_csv->skiprows = args->skiprows;
	raw_csv->skipfooter = args->skipfooter;
	raw_csv->nrows = args->nrows;
	raw_csv->prefix = args->prefix == nullptr ? "" : string(args->prefix);

	if (args->delim_whitespace) {
		raw_csv->opts.delimiter = ' ';
		raw_csv->opts.multi_delimiter = true;
	} else {
		raw_csv->opts.delimiter = args->delimiter;
		raw_csv->opts.multi_delimiter = false;
	}
	if (args->windowslinetermination) {
		raw_csv->opts.terminator = '\n';
	} else {
		raw_csv->opts.terminator = args->lineterminator;
	}
	if (args->quotechar != '\0') {
		raw_csv->opts.quotechar = args->quotechar;
		raw_csv->opts.keepquotes = !args->quoting;
		raw_csv->opts.doublequote = args->doublequote;
	} else {
		raw_csv->opts.quotechar = args->quotechar;
		raw_csv->opts.keepquotes = true;
		raw_csv->opts.doublequote = false;
	}
	raw_csv->opts.skipblanklines = args->skip_blank_lines;
	raw_csv->opts.comment = args->comment;
	raw_csv->opts.dayfirst = args->dayfirst;
	raw_csv->opts.decimal = args->decimal;
	raw_csv->opts.thousands = args->thousands;
	if (raw_csv->opts.decimal == raw_csv->opts.delimiter) {
		checkError(GDF_INVALID_API_CALL, "Decimal point cannot be the same as the delimiter");
	}
	if (raw_csv->opts.thousands == raw_csv->opts.delimiter) {
		checkError(GDF_INVALID_API_CALL, "Thousands separator cannot be the same as the delimiter");
	}

	string compression_type;
	error = inferCompressionType(args->compression, args->filepath_or_buffer, compression_type);
	checkError(error, "call to inferCompressionType");

	raw_csv->byte_range_offset = args->byte_range_offset;
	raw_csv->byte_range_size = args->byte_range_size;
	if (raw_csv->byte_range_offset > 0 || raw_csv->byte_range_size > 0) {
		if (raw_csv->nrows >= 0 || raw_csv->skiprows > 0 || raw_csv->skipfooter > 0) {
			checkError(GDF_INVALID_API_CALL, 
				"Cannot manually limit rows to be read when using the byte range parameter");
		}
		if (compression_type != "none") {
			checkError(GDF_INVALID_API_CALL, 
				"Cannot read compressed input when using the byte range parameter");
		}
	}

	// Handle user-defined booleans values, whereby field data is substituted
	// with true/false values; CUDF booleans are int types of 0 or 1
	// The true/false value strings are converted to integers which are used
	// by the data conversion kernel for comparison and value replacement
	if ((args->true_values != NULL) && (args->num_true_values > 0)) {
		thrust::host_vector<int32_t> h_values(args->num_true_values);
		for (int i = 0; i < args->num_true_values; ++i) {
			h_values[i] = convertStrToValue<int32_t>(args->true_values[i], 0, strlen(args->true_values[i]) - 1, raw_csv->opts);
		}
		raw_csv->d_trueValues = h_values;
	}
	if ((args->false_values != NULL) && (args->num_false_values > 0)) {
		thrust::host_vector<int32_t> h_values(args->num_false_values);
		for (int i = 0; i < args->num_false_values; ++i) {
			h_values[i] = convertStrToValue<int32_t>(args->false_values[i], 0, strlen(args->false_values[i]) - 1, raw_csv->opts);
		}
		raw_csv->d_falseValues = h_values;
	}

	if (args->na_filter && 
		(args->keep_default_na || (args->na_values != nullptr && args->num_na_values > 0))) {
		vector<string> na_values{
			"#N/A", "#N/A N/A", "#NA", "-1.#IND", 
			"-1.#QNAN", "-NaN", "-nan", "1.#IND", 
			"1.#QNAN", "N/A", "NA", "NULL", 
			"NaN", "n/a", "nan", "null"};
		if(!args->keep_default_na){
			na_values.clear();
		}

		if (args->na_values != nullptr && args->num_na_values > 0) {
			for (int i = 0; i < args->num_na_values; ++i) {
				na_values.emplace_back(args->na_values[i]);
			}
		}

		raw_csv->d_naTrie = createSerializedTrie(na_values);
	}

	raw_csv->opts.trueValues       = raw_csv->d_trueValues.data().get();
	raw_csv->opts.trueValuesCount  = raw_csv->d_trueValues.size();
	raw_csv->opts.falseValues      = raw_csv->d_falseValues.data().get();
	raw_csv->opts.falseValuesCount = raw_csv->d_falseValues.size();

	//-----------------------------------------------------------------------------
	// memory map in the data
	void * 	map_data = NULL;
	size_t	map_size = 0;
	size_t	map_offset = 0;
	int fd = 0;
	if (args->input_data_form == gdf_csv_input_form::FILE_PATH)
	{
		fd = open(args->filepath_or_buffer, O_RDONLY );
		if (fd < 0) 		{ close(fd); checkError(GDF_FILE_ERROR, "Error opening file"); }

		struct stat st{};
		if (fstat(fd, &st)) { close(fd); checkError(GDF_FILE_ERROR, "cannot stat file");   }
	
		const auto file_size = st.st_size;
		const auto page_size = sysconf(_SC_PAGESIZE);

		if (args->byte_range_offset >= (size_t)file_size) { 
			close(fd); 
			checkError(GDF_INVALID_API_CALL, "The byte_range offset is larger than the file size");
		}

		// Have to align map offset to page size
		map_offset = (args->byte_range_offset/page_size)*page_size;

		// Set to rest-of-the-file size, will reduce based on the byte range size
		raw_csv->num_bytes = map_size = file_size - map_offset;

		// Include the page padding in the mapped size
		const size_t page_padding = args->byte_range_offset - map_offset;
		const size_t padded_byte_range_size = raw_csv->byte_range_size + page_padding;

		if (raw_csv->byte_range_size != 0 && padded_byte_range_size < map_size) {
			// Need to make sure that w/ padding we don't overshoot the end of file
			map_size = min(padded_byte_range_size + calculateMaxRowSize(args->num_cols), map_size);
			// Ignore page padding for parsing purposes
			raw_csv->num_bytes = map_size - page_padding;
		}

		map_data = mmap(0, map_size, PROT_READ, MAP_PRIVATE, fd, map_offset);
	
		if (map_data == MAP_FAILED || map_size==0) { close(fd); checkError(GDF_C_ERROR, "Error mapping file"); }
	}
	else if (args->input_data_form == gdf_csv_input_form::HOST_BUFFER)
	{
		map_data = (void *)args->filepath_or_buffer;
		raw_csv->num_bytes = map_size = args->buffer_size;
	}
	else { checkError(GDF_C_ERROR, "invalid input type"); }

	const char* h_uncomp_data;
	size_t h_uncomp_size = 0;
	// Used when the input data is compressed, to ensure the allocated uncompressed data is freed
	vector<char> h_uncomp_data_owner;
	if (compression_type == "none") {
		// Do not use the owner vector here to avoid copying the whole file to the heap
		h_uncomp_data = (const char*)map_data + (args->byte_range_offset - map_offset);
		h_uncomp_size = raw_csv->num_bytes;
	}
	else {
		error = getUncompressedHostData( (const char *)map_data, map_size, compression_type, h_uncomp_data_owner);
		checkError(error, "call to getUncompressedHostData");
		h_uncomp_data = h_uncomp_data_owner.data();
		h_uncomp_size = h_uncomp_data_owner.size();
	}
	assert(h_uncomp_data != nullptr);
	assert(h_uncomp_size != 0);

	error = launch_countRecords(h_uncomp_data, h_uncomp_size, raw_csv, raw_csv->num_records);
	checkError(error, "call to record number of rows");

	//-----------------------------------------------------------------------------
	//-- Allocate space to hold the record starting point
	RMM_TRY( RMM_ALLOC(&raw_csv->recStart, sizeof(cu_recstart_t) * raw_csv->num_records, 0) ); 

	//-----------------------------------------------------------------------------
	//-- Scan data and set the starting positions
	error = launch_storeRecordStart(h_uncomp_data, h_uncomp_size, raw_csv);
	checkError(error, "call to record initial position store");

	// Previous kernel stores the record pinput_file.typeositions as encountered by all threads
	// Sort the record positions as subsequent processing may require filtering
	// certain rows or other processing on specific records
	thrust::sort(rmm::exec_policy()->on(0), raw_csv->recStart, raw_csv->recStart + raw_csv->num_records);

	// Currently, ignoring lineterminations within quotes is handled by recording
	// the records of both, and then filtering out the records that is a quotechar
	// or a linetermination within a quotechar pair. The future major refactoring
	// of csv_reader and its kernels will probably use a different tactic.
	if (raw_csv->opts.quotechar != '\0') {
		vector<cu_recstart_t> h_rec_starts(raw_csv->num_records);
		const size_t rec_start_size = sizeof(cu_recstart_t) * (h_rec_starts.size());
		CUDA_TRY( hipMemcpy(h_rec_starts.data(), raw_csv->recStart, rec_start_size, hipMemcpyDeviceToHost) );

		auto recCount = raw_csv->num_records;

		bool quotation = false;
		for (gdf_size_type i = 1; i < raw_csv->num_records; ++i) {
			if (h_uncomp_data[h_rec_starts[i] - 1] == raw_csv->opts.quotechar) {
				quotation = !quotation;
				h_rec_starts[i] = raw_csv->num_bytes;
				recCount--;
			}
			else if (quotation) {
				h_rec_starts[i] = raw_csv->num_bytes;
				recCount--;
			}
		}

		CUDA_TRY( hipMemcpy(raw_csv->recStart, h_rec_starts.data(), rec_start_size, hipMemcpyHostToDevice) );
		thrust::sort(rmm::exec_policy()->on(0), raw_csv->recStart, raw_csv->recStart + raw_csv->num_records);
		raw_csv->num_records = recCount;
	}

	error = uploadDataToDevice(h_uncomp_data, h_uncomp_size, raw_csv);
	if (error != GDF_SUCCESS) {
		return error;
	}

	//-----------------------------------------------------------------------------
	//-- Populate the header

	// Check if the user gave us a list of column names
	if(args->names == nullptr) {

		error = setColumnNamesFromCsv(raw_csv);
		if (error != GDF_SUCCESS) {
			return error;
		}
		const int h_num_cols = raw_csv->col_names.size();

		// Allocating a boolean array that will use to state if a column needs to read or filtered.
		raw_csv->h_parseCol = (bool*)malloc(sizeof(bool) * (h_num_cols));
		RMM_TRY( RMM_ALLOC((void**)&raw_csv->d_parseCol,(sizeof(bool) * (h_num_cols)),0 ) );
		for (int i = 0; i<h_num_cols; i++)
			raw_csv->h_parseCol[i]=true;
		
		// Rename empty column names to "Unnamed: col_index"
		for (size_t col_idx = 0; col_idx < raw_csv->col_names.size(); ++col_idx) {
			if (raw_csv->col_names[col_idx].empty()) {
				raw_csv->col_names[col_idx] = string("Unnamed: ") + std::to_string(col_idx);
			}
		}

		int h_dup_cols_removed = 0;
		// Looking for duplicates
		for (auto it = raw_csv->col_names.begin(); it != raw_csv->col_names.end(); it++){
			bool found_dupe = false;
			for (auto it2 = (it+1); it2 != raw_csv->col_names.end(); it2++){
				if (*it==*it2){
					found_dupe=true;
					break;
				}
			}
			if(found_dupe){
				int count=1;
				for (auto it2 = (it+1); it2 != raw_csv->col_names.end(); it2++){
					if (*it==*it2){
						if(args->mangle_dupe_cols){
							// Replace all the duplicates of column X with X.1,X.2,... First appearance stays as X.
							std::string newColName  = *it2;
							newColName += "." + std::to_string(count); 
							count++;
							*it2 = newColName;							
						} else{
							// All duplicate fields will be ignored.
							int pos=std::distance(raw_csv->col_names.begin(), it2);
							raw_csv->h_parseCol[pos]=false;
							h_dup_cols_removed++;
						}
					}
				}
			}
		}

		raw_csv->num_actual_cols = h_num_cols;							// Actual number of columns in the CSV file
		raw_csv->num_active_cols = h_num_cols-h_dup_cols_removed;		// Number of fields that need to be processed based on duplicatation fields

		CUDA_TRY(hipMemcpy(raw_csv->d_parseCol, raw_csv->h_parseCol, sizeof(bool) * (h_num_cols), hipMemcpyHostToDevice));
	}
	else {
		raw_csv->h_parseCol = (bool*)malloc(sizeof(bool) * (args->num_cols));
		RMM_TRY( RMM_ALLOC((void**)&raw_csv->d_parseCol,(sizeof(bool) * (args->num_cols)),0 ) );

		for (int i = 0; i<raw_csv->num_actual_cols; i++){
			raw_csv->h_parseCol[i]=true;
			std::string col_name 	= args->names[i];
			raw_csv->col_names.push_back(col_name);

		}
		CUDA_TRY(hipMemcpy(raw_csv->d_parseCol, raw_csv->h_parseCol, sizeof(bool) * (args->num_cols), hipMemcpyHostToDevice));
	}

	// User can give
	if (args->use_cols_int!=NULL || args->use_cols_char!=NULL){
		if(args->use_cols_int!=NULL){
			for (int i = 0; i<raw_csv->num_actual_cols; i++)
				raw_csv->h_parseCol[i]=false;
			for(int i=0; i < args->use_cols_int_len; i++){
				int pos = args->use_cols_int[i];
				raw_csv->h_parseCol[pos]=true;
			}
			raw_csv->num_active_cols = args->use_cols_int_len;
		}else{
			for (int i = 0; i<raw_csv->num_actual_cols; i++)
				raw_csv->h_parseCol[i]=false;
			int countFound=0;
			for(int i=0; i < args->use_cols_char_len; i++){
				std::string colName(args->use_cols_char[i]);
				for (auto it = raw_csv->col_names.begin(); it != raw_csv->col_names.end(); it++){
					if(colName==*it){
						countFound++;
						int pos=std::distance(raw_csv->col_names.begin(), it);
						raw_csv->h_parseCol[pos]=true;
						break;
					}
				}
			}
			raw_csv->num_active_cols = countFound;
		}
		CUDA_TRY(hipMemcpy(raw_csv->d_parseCol, raw_csv->h_parseCol, sizeof(bool) * (raw_csv->num_actual_cols), hipMemcpyHostToDevice));
	}


	//-----------------------------------------------------------------------------
	//---  done with host data
	if (args->input_data_form == gdf_csv_input_form::FILE_PATH)
	{
		close(fd);
		munmap(map_data, map_size);
	}


	//-----------------------------------------------------------------------------
	//--- Auto detect types of the vectors

	if(args->dtype==NULL){
		if (raw_csv->num_records == 0) {
			checkError(GDF_INVALID_API_CALL, "read_csv: no data available for data type inference");
		}

		column_data_t *d_ColumnData,*h_ColumnData;

		h_ColumnData = (column_data_t*)malloc(sizeof(column_data_t) * (raw_csv->num_active_cols));
		RMM_TRY( RMM_ALLOC((void**)&d_ColumnData,(sizeof(column_data_t) * (raw_csv->num_active_cols)),0 ) );

		CUDA_TRY( hipMemset(d_ColumnData,	0, 	(sizeof(column_data_t) * (raw_csv->num_active_cols)) ) ) ;

		launch_dataTypeDetection(raw_csv, d_ColumnData);

		CUDA_TRY( hipMemcpy(h_ColumnData,d_ColumnData, sizeof(column_data_t) * (raw_csv->num_active_cols), hipMemcpyDeviceToHost));

	    vector<gdf_dtype>	d_detectedTypes;			// host: array of dtypes (since gdf_columns are not created until end)

		raw_csv->dtypes.clear();

		for(int col = 0; col < raw_csv->num_active_cols; col++){
			unsigned long long countInt = h_ColumnData[col].countInt8+h_ColumnData[col].countInt16+
										  h_ColumnData[col].countInt32+h_ColumnData[col].countInt64;

			if (h_ColumnData[col].countNULL == raw_csv->num_records){
				d_detectedTypes.push_back(GDF_INT8); // Entire column is NULL. Allocating the smallest amount of memory
			} else if(h_ColumnData[col].countString>0L){
				d_detectedTypes.push_back(GDF_CATEGORY); // For auto-detection, we are currently not supporting strings.
			} else if(h_ColumnData[col].countDateAndTime>0L){
				d_detectedTypes.push_back(GDF_DATE64);
			} else if(h_ColumnData[col].countFloat > 0L  ||  
				(h_ColumnData[col].countFloat==0L && countInt >0L && h_ColumnData[col].countNULL >0L) ) {
				// The second condition has been added to conform to PANDAS which states that a colum of 
				// integers with a single NULL record need to be treated as floats.
				d_detectedTypes.push_back(GDF_FLOAT64);
			}
			else { 
				d_detectedTypes.push_back(GDF_INT64);
			}
		}

		raw_csv->dtypes=d_detectedTypes;

		free(h_ColumnData);
		RMM_TRY( RMM_FREE( d_ColumnData, 0 ) );
	}
	else{
		for ( int x = 0; x < raw_csv->num_actual_cols; x++) {

			std::string temp_type 	= args->dtype[x];
			gdf_dtype col_dtype		= convertStringToDtype( temp_type );

			if (col_dtype == GDF_invalid)
				return GDF_UNSUPPORTED_DTYPE;

			raw_csv->dtypes.push_back(col_dtype);
		}
	}


	//-----------------------------------------------------------------------------
	//--- allocate space for the results
	gdf_column **cols = (gdf_column **)malloc( sizeof(gdf_column *) * raw_csv->num_active_cols);

	void **d_data,**h_data;
	gdf_valid_type **d_valid,**h_valid;
    unsigned long long	*d_valid_count;
	gdf_dtype *d_dtypes,*h_dtypes;





	h_dtypes 		= (gdf_dtype*)malloc (	sizeof(gdf_dtype)* (raw_csv->num_active_cols));
	h_data 			= (void**)malloc (	sizeof(void*)* (raw_csv->num_active_cols));
	h_valid 		= (gdf_valid_type**)malloc (	sizeof(gdf_valid_type*)* (raw_csv->num_active_cols));

	RMM_TRY( RMM_ALLOC((void**)&d_dtypes, 		(sizeof(gdf_dtype) 			* raw_csv->num_active_cols), 0 ) );
	RMM_TRY( RMM_ALLOC((void**)&d_data, 		(sizeof(void *)				* raw_csv->num_active_cols), 0 ) );
	RMM_TRY( RMM_ALLOC((void**)&d_valid, 		(sizeof(gdf_valid_type *)	* raw_csv->num_active_cols), 0 ) );
	RMM_TRY( RMM_ALLOC((void**)&d_valid_count, 	(sizeof(unsigned long long) * raw_csv->num_active_cols), 0 ) );
	CUDA_TRY( hipMemset(d_valid_count,	0, 		(sizeof(unsigned long long)	* raw_csv->num_active_cols)) );


	int stringColCount=0;
	for (int col = 0; col < raw_csv->num_active_cols; col++) {
		if(raw_csv->dtypes[col]==gdf_dtype::GDF_STRING)
			stringColCount++;
	}

	string_pair **h_str_cols = NULL, **d_str_cols = NULL;

	if (stringColCount > 0 ) {
		h_str_cols = (string_pair**) malloc ((sizeof(string_pair *)	* stringColCount));
		RMM_TRY( RMM_ALLOC((void**)&d_str_cols, 	(sizeof(string_pair *)		* stringColCount), 0) );

		for (int col = 0; col < stringColCount; col++) {
			RMM_TRY( RMM_ALLOC((void**)(h_str_cols + col), sizeof(string_pair) * (raw_csv->num_records), 0) );
		}

		CUDA_TRY(hipMemcpy(d_str_cols, h_str_cols, sizeof(string_pair *)	* stringColCount, hipMemcpyHostToDevice));
	}

	for (int acol = 0,col=-1; acol < raw_csv->num_actual_cols; acol++) {
		if(raw_csv->h_parseCol[acol]==false)
			continue;
		col++;

		gdf_column *gdf = (gdf_column *)malloc(sizeof(gdf_column) * 1);

		gdf->size		= raw_csv->num_records;
		gdf->dtype		= raw_csv->dtypes[col];
		gdf->null_count	= 0;						// will be filled in later

		//--- column name
		std::string str = raw_csv->col_names[acol];
		int len = str.length() + 1;
		gdf->col_name = (char *)malloc(sizeof(char) * len);
		memcpy(gdf->col_name, str.c_str(), len);
		gdf->col_name[len -1] = '\0';

		allocateGdfDataSpace(gdf);

		cols[col] 		= gdf;
		h_dtypes[col] 	= gdf->dtype;
		h_data[col] 	= gdf->data;
		h_valid[col] 	= gdf->valid;	
    }

	CUDA_TRY( hipMemcpy(d_dtypes,h_dtypes, sizeof(gdf_dtype) * (raw_csv->num_active_cols), hipMemcpyHostToDevice));
	CUDA_TRY( hipMemcpy(d_data,h_data, sizeof(void*) * (raw_csv->num_active_cols), hipMemcpyHostToDevice));
	CUDA_TRY( hipMemcpy(d_valid,h_valid, sizeof(gdf_valid_type*) * (raw_csv->num_active_cols), hipMemcpyHostToDevice));

	free(h_dtypes); 
	free(h_valid); 
	free(h_data); 

	if (raw_csv->num_records != 0) {
		error = launch_dataConvertColumns(raw_csv, d_data, d_valid, d_dtypes, d_str_cols, d_valid_count);
		if (error != GDF_SUCCESS) {
			return error;
		}
		// Sync with the default stream, just in case create_from_index() is asynchronous 
		hipStreamSynchronize(0);

		stringColCount=0;
		for (int col = 0; col < raw_csv->num_active_cols; col++) {

			gdf_column *gdf = cols[col];

			if (gdf->dtype != gdf_dtype::GDF_STRING)
				continue;

			NVStrings* const stringCol = NVStrings::create_from_index(h_str_cols[stringColCount],size_t(raw_csv->num_records));
			if ((raw_csv->opts.quotechar != '\0') && (raw_csv->opts.doublequote==true)) {
				// In PANDAS, default of enabling doublequote for two consecutive
				// quotechar in quote fields results in reduction to single
				const string quotechar(1, raw_csv->opts.quotechar);
				const string doublequotechar(2, raw_csv->opts.quotechar);
				gdf->data = stringCol->replace(doublequotechar.c_str(), quotechar.c_str());
				NVStrings::destroy(stringCol);
			}
			else {
				gdf->data = stringCol;
			}

			RMM_TRY( RMM_FREE( h_str_cols [stringColCount], 0 ) );

			stringColCount++;
		}

		vector<unsigned long long>	h_valid_count(raw_csv->num_active_cols);
		CUDA_TRY( hipMemcpy(h_valid_count.data(), d_valid_count, sizeof(unsigned long long) * h_valid_count.size(), hipMemcpyDeviceToHost));

		//--- set the null count
		for (size_t col = 0; col < h_valid_count.size(); col++) {
			cols[col]->null_count = raw_csv->num_records - h_valid_count[col];
		}
	}

	// free up space that is no longer needed
	if (h_str_cols != NULL)
		free ( h_str_cols);

	free(raw_csv->h_parseCol);

	if (d_str_cols != NULL)
		RMM_TRY( RMM_FREE( d_str_cols, 0 ) ); 

	RMM_TRY( RMM_FREE( d_valid, 0 ) );
	RMM_TRY( RMM_FREE( d_valid_count, 0 ) );
	RMM_TRY( RMM_FREE( d_dtypes, 0 ) );
	RMM_TRY( RMM_FREE( d_data, 0 ) ); 

	RMM_TRY( RMM_FREE( raw_csv->recStart, 0 ) ); 
	RMM_TRY( RMM_FREE( raw_csv->d_parseCol, 0 ) ); 
	RMM_TRY( RMM_FREE ( raw_csv->data, 0) );


	args->data 			= cols;
	args->num_cols_out	= raw_csv->num_active_cols;
	args->num_rows_out	= raw_csv->num_records;

	delete raw_csv;
	return error;
}



/*
 * What is passed in is the data type as a string, need to convert that into gdf_dtype enum
 */
gdf_dtype convertStringToDtype(std::string &dtype) {

	if (dtype.compare( "str") == 0) 		return GDF_STRING;
	if (dtype.compare( "date") == 0) 		return GDF_DATE64;
	if (dtype.compare( "date32") == 0) 		return GDF_DATE32;
	if (dtype.compare( "date64") == 0) 		return GDF_DATE64;
	if (dtype.compare( "timestamp") == 0)	return GDF_TIMESTAMP;
	if (dtype.compare( "category") == 0) 	return GDF_CATEGORY;
	if (dtype.compare( "float") == 0)		return GDF_FLOAT32;
	if (dtype.compare( "float32") == 0)		return GDF_FLOAT32;
	if (dtype.compare( "float64") == 0)		return GDF_FLOAT64;
	if (dtype.compare( "double") == 0)		return GDF_FLOAT64;
	if (dtype.compare( "short") == 0)		return GDF_INT16;
	if (dtype.compare( "int") == 0)			return GDF_INT32;
	if (dtype.compare( "int32") == 0)		return GDF_INT32;
	if (dtype.compare( "int64") == 0)		return GDF_INT64;
	if (dtype.compare( "long") == 0)		return GDF_INT64;

	return GDF_invalid;
}


/**---------------------------------------------------------------------------*
 * @brief Infer the compression type from the compression parameter and 
 * the input file name
 * 
 * Returns "none" if the input is not compressed.
 * 
 * @param[in] compression_arg Input string that is potentially describing 
 * the compression type. Can also be nullptr, "none", or "infer"
 * @param[in] filepath path + name of the input file
 * @param[out] compression_type String describing the inferred compression type
 * 
 * @return gdf_error with error code on failure, otherwise GDF_SUCCESS
 *---------------------------------------------------------------------------**/
gdf_error inferCompressionType(const char* compression_arg, const char* filepath, string& compression_type)
{
	if (compression_arg && 0 == strcasecmp(compression_arg, "none")) {
		compression_arg = nullptr;
	}
	if (compression_arg && 0 == strcasecmp(compression_arg, "infer"))
	{
		const char *file_ext = strrchr(filepath, '.');
		compression_arg = nullptr;
		if (file_ext)
		{
			if (!strcasecmp(file_ext, ".gz"))
				compression_arg = "gzip";
			else if (!strcasecmp(file_ext, ".zip"))
				compression_arg = "zip";
			else if (!strcasecmp(file_ext, ".bz2"))
				compression_arg = "bz2";
			else if (!strcasecmp(file_ext, ".xz"))
				compression_arg = "xz";
			else {
				// TODO: return error here
			}
		}
	}
	compression_type = compression_arg == nullptr? "none":string(compression_arg);
	
	return GDF_SUCCESS;
}


/**---------------------------------------------------------------------------*
 * @brief Uncompresses the input data and stores the allocated result into 
 * a vector.
 * 
 * @param[in] h_data Pointer to the csv data in host memory
 * @param[in] num_bytes Size of the input data, in bytes
 * @param[in] compression String describing the compression type
 * @param[out] h_uncomp_data Vector containing the output uncompressed data
 * 
 * @return gdf_error with error code on failure, otherwise GDF_SUCCESS
 *---------------------------------------------------------------------------**/
gdf_error getUncompressedHostData(const char* h_data, size_t num_bytes, const string& compression, vector<char>& h_uncomp_data) 
{	
	int comp_type = IO_UNCOMP_STREAM_TYPE_INFER;
	if (compression == "gzip")
		comp_type = IO_UNCOMP_STREAM_TYPE_GZIP;
	else if (compression == "zip")
		comp_type = IO_UNCOMP_STREAM_TYPE_ZIP;
	else if (compression == "bz2")
		comp_type = IO_UNCOMP_STREAM_TYPE_BZIP2;
	else if (compression == "xz")
		comp_type = IO_UNCOMP_STREAM_TYPE_XZ;

	return io_uncompress_single_h2d(h_data, num_bytes, comp_type, h_uncomp_data);
}


/**---------------------------------------------------------------------------*
 * @brief Uploads the relevant segment of the input csv data onto the GPU.
 * 
 * Only rows that need to be read are copied to the GPU, based on parameters
 * like nrows, skipheader, skipfooter.
 * Also updates the array of record starts to match the device data offset.
 * 
 * @param[in] h_uncomp_data Pointer to the uncompressed csv data in host memory
 * @param[in] h_uncomp_size Size of the input data, in bytes
 * @param[in,out] raw_csv Structure containing the csv parsing parameters
 * and intermediate results
 * 
 * @return gdf_error with error code on failure, otherwise GDF_SUCCESS
 *---------------------------------------------------------------------------**/
gdf_error uploadDataToDevice(const char *h_uncomp_data, size_t h_uncomp_size,
                             raw_csv_t *raw_csv) {

  // Exclude the rows that are to be skipped from the start
  GDF_REQUIRE(raw_csv->num_records > raw_csv->skiprows, GDF_INVALID_API_CALL);
  const auto first_row = raw_csv->skiprows;
  raw_csv->num_records = raw_csv->num_records - first_row;

  std::vector<cu_recstart_t> h_rec_starts(raw_csv->num_records);
  CUDA_TRY(hipMemcpy(h_rec_starts.data(), raw_csv->recStart + first_row,
                      sizeof(cu_recstart_t) * h_rec_starts.size(),
                      hipMemcpyDefault));

  // Trim lines that are outside range, but keep one greater for the end offset
  if (raw_csv->byte_range_size != 0) {
    auto it = h_rec_starts.end() - 1;
    while (it >= h_rec_starts.begin() &&
           *it > cu_recstart_t(raw_csv->byte_range_size)) {
      --it;
    }
    if ((it + 2) < h_rec_starts.end()) {
      h_rec_starts.erase(it + 2, h_rec_starts.end());
    }
  }

  // Discard only blank lines, only fully comment lines, or both.
  // If only handling one of them, ensure it doesn't match against \0 as we do
  // not want certain scenarios to be filtered out (end-of-file)
  if (raw_csv->opts.skipblanklines || raw_csv->opts.comment != '\0') {
    const auto match1 = raw_csv->opts.skipblanklines ? raw_csv->opts.terminator
                                                     : raw_csv->opts.comment;
    const auto match2 = raw_csv->opts.comment != '\0' ? raw_csv->opts.comment
                                                      : match1;
    h_rec_starts.erase(
        std::remove_if(h_rec_starts.begin(), h_rec_starts.end(),
                       [&](cu_recstart_t i) {
                         return (h_uncomp_data[i] == match1 ||
                                 h_uncomp_data[i] == match2);
                       }),
        h_rec_starts.end());
  }

  raw_csv->num_records = h_rec_starts.size();

  // Exclude the rows before the header row (inclusive)
  // But copy the header data for parsing the column names later (if necessary)
  if (raw_csv->header_row >= 0) {
    raw_csv->header.assign(
        h_uncomp_data + h_rec_starts[raw_csv->header_row],
        h_uncomp_data + h_rec_starts[raw_csv->header_row + 1]);
    h_rec_starts.erase(h_rec_starts.begin(),
                       h_rec_starts.begin() + raw_csv->header_row + 1);
    raw_csv->num_records = h_rec_starts.size();
  }

  // Exclude the rows that exceed past the requested number
  if (raw_csv->nrows >= 0 && raw_csv->nrows < raw_csv->num_records) {
    h_rec_starts.resize(raw_csv->nrows + 1);    // include end offset
    raw_csv->num_records = h_rec_starts.size();
  }

  // Exclude the rows that are to be skipped from the end
  if (raw_csv->skipfooter > 0) {
    h_rec_starts.resize(h_rec_starts.size() - raw_csv->skipfooter);
    raw_csv->num_records = h_rec_starts.size();
  }

  // Check that there is actual data to parse
  GDF_REQUIRE(raw_csv->num_records > 0, GDF_INVALID_API_CALL);

  const auto start_offset = h_rec_starts.front();
  const auto end_offset = h_rec_starts.back();
  raw_csv->num_bytes = end_offset - start_offset;
  assert(raw_csv->num_bytes <= h_uncomp_size);
  raw_csv->num_bits = (raw_csv->num_bytes + 63) / 64;

  // Resize and upload the rows of interest
  RMM_TRY(RMM_REALLOC(&raw_csv->recStart,
                      sizeof(cu_recstart_t) * raw_csv->num_records, 0));
  CUDA_TRY(hipMemcpy(raw_csv->recStart, h_rec_starts.data(),
                      sizeof(cu_recstart_t) * raw_csv->num_records,
                      hipMemcpyDefault));

  // Upload the raw data that is within the rows of interest
  RMM_TRY(RMM_ALLOC(&raw_csv->data, raw_csv->num_bytes, 0));
  CUDA_TRY(hipMemcpy(raw_csv->data, h_uncomp_data + start_offset,
                      raw_csv->num_bytes, hipMemcpyHostToDevice));

  // Adjust row start positions to account for the data subcopy
  thrust::transform(rmm::exec_policy()->on(0), raw_csv->recStart,
                    raw_csv->recStart + raw_csv->num_records,
                    thrust::make_constant_iterator(start_offset),
                    raw_csv->recStart, thrust::minus<cu_recstart_t>());

  // The array of row offsets includes EOF
  // reduce the number of records by one to exclude it from the row count
  raw_csv->num_records--;

  return GDF_SUCCESS;
}


/**---------------------------------------------------------------------------*
 * @brief Allocates memory for a column's parsed output and its validity bitmap
 *
 * Memory for column data is simply based upon number of rows and the size of
 * the output data type, regardless of actual validity of the row element.
 *
 * @param[in,out] col The column whose memory will be allocated
 *
 * @return gdf_error GDF_SUCCESS upon completion
 *---------------------------------------------------------------------------**/
gdf_error allocateGdfDataSpace(gdf_column *col) {
  // TODO: We should not need to allocate space if there is nothing to parse
  // Need to debug/refactor the code to eliminate this requirement
  const auto num_rows = std::max(col->size, 1);
  const auto num_masks = gdf_get_num_chars_bitmask(num_rows);

  RMM_TRY(RMM_ALLOC(&col->valid, sizeof(gdf_valid_type) * num_masks, 0));
  CUDA_TRY(hipMemset(col->valid, 0, sizeof(gdf_valid_type) * num_masks));

  if (col->dtype != gdf_dtype::GDF_STRING) {
    int column_byte_width = 0;
    checkError(get_column_byte_width(col, &column_byte_width),
               "Could not get column width using data type");
    RMM_TRY(RMM_ALLOC(&col->data, num_rows * column_byte_width, 0));
  }

  return GDF_SUCCESS;
}

//----------------------------------------------------------------------------------------------------------------
//				CUDA Kernels
//----------------------------------------------------------------------------------------------------------------


/**---------------------------------------------------------------------------*
 * @brief Counts the number of rows in the input csv file.
 * 
 * Does not load the entire file into the GPU memory at any time, so it can 
 * be used to parse large files.
 * Does not take quotes into consideration, so it will return extra rows
 * if the line terminating characters are present within quotes.
 * Because of this the result should be postprocessed to remove 
 * the fake line endings.
 * 
 * @param[in] h_data Pointer to the csv data in host memory
 * @param[in] h_size Size of the input data, in bytes
 * @param[in] terminator Line terminator character
 * @param[in] quote Quote character
 * @param[out] rec_cnt The resulting number of rows (records)
 * 
 * @return gdf_error with error code on failure, otherwise GDF_SUCCESS
 *---------------------------------------------------------------------------**/
gdf_error launch_countRecords(const char *h_data, size_t h_size,
                              raw_csv_t *raw_csv, gdf_size_type &rec_cnt)
{
	const size_t chunk_count = (h_size + max_chunk_bytes - 1) / max_chunk_bytes;
	rmm::device_vector<cu_reccnt_t> d_counts(chunk_count);

	char* d_chunk = nullptr;
	RMM_TRY(RMM_ALLOC (&d_chunk, max_chunk_bytes, 0)); 

	int blockSize;		// suggested thread count to use
	int minGridSize;	// minimum block count required
	CUDA_TRY(hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, countRecords));

	for (size_t ci = 0; ci < chunk_count; ++ci) {
		const auto h_chunk = h_data + ci * max_chunk_bytes;
		const auto chunk_bytes = std::min((size_t)(h_size - ci * max_chunk_bytes), max_chunk_bytes);
		const auto chunk_bits = (chunk_bytes + 63) / 64;

		// Copy chunk to device
		CUDA_TRY(hipMemcpy(d_chunk, h_chunk, chunk_bytes, hipMemcpyDefault));

		const int gridSize = (chunk_bits + blockSize - 1) / blockSize;
		countRecords <<< gridSize, blockSize >>> (
			d_chunk, raw_csv->opts.terminator, raw_csv->opts.quotechar,
			chunk_bytes, chunk_bits, thrust::raw_pointer_cast(&d_counts[ci])
			);
	}

	RMM_TRY( RMM_FREE(d_chunk, 0) );

	CUDA_TRY(hipGetLastError());

	// Row count is used to allocate/track row start positions
	// If not starting at an offset, add an extra row to account for offset=0
	rec_cnt = thrust::reduce(rmm::exec_policy()->on(0), d_counts.begin(), d_counts.end());
	if (raw_csv->byte_range_offset == 0) {
		rec_cnt++;
	}

	return GDF_SUCCESS;
}


/**---------------------------------------------------------------------------* 
 * @brief CUDA kernel that counts the number of rows in the given 
 * file segment, based on the location of line terminators. 
 * 
 * @param[in] data Device memory pointer to the csv data, 
 * potentially a chunk of the whole file
 * @param[in] terminator Line terminator character
 * @param[in] quotechar Quote character
 * @param[in] num_bytes Number of bytes in the input data
 * @param[in] num_bits Number of 'bits' in the input data. Each 'bit' is
 * processed by a separate CUDA thread
 * @param[in,out] num_records Device memory pointer to the number of found rows
 * 
 * @return gdf_error with error code on failure, otherwise GDF_SUCCESS
 *---------------------------------------------------------------------------**/
__global__ void countRecords(char *data, const char terminator, const char quotechar, long num_bytes, long num_bits, 
	cu_reccnt_t* num_records) {

	// thread IDs range per block, so also need the block id
	const long tid = threadIdx.x + (blockDim.x * blockIdx.x);

	if (tid >= num_bits)
		return;

	// data ID is a multiple of 64
	const long did = tid * 64L;

	const char *raw = (data + did);

	const long byteToProcess = ((did + 64L) < num_bytes) ? 64L : (num_bytes - did);

	// process the data
	cu_reccnt_t tokenCount = 0;
	for (long x = 0; x < byteToProcess; x++) {
		
		// Scan and log records. If quotations are enabled, then also log quotes
		// for a postprocess ignore, as the chunk here has limited visibility.
		if ((raw[x] == terminator) || (quotechar != '\0' && raw[x] == quotechar)) {
			tokenCount++;
		} else if (terminator == '\n' && (x + 1L) < byteToProcess && 
		           raw[x] == '\r' && raw[x + 1L] == '\n') {
			x++;
			tokenCount++;
		}

	}
	atomicAdd(num_records, tokenCount);
}


/**---------------------------------------------------------------------------*
 * @brief Finds the start of each row (record) in the given file, based on
 * the location of line terminators. The offset of each found row is stored 
 * in the recStart data member of the csvData parameter.
 * 
 * Does not load the entire file into the GPU memory at any time, so it can 
 * be used to parse large files.
 * Does not take quotes into consideration, so it will return extra rows
 * if the line terminating characters are present within quotes.
 * Because of this the result should be postprocessed to remove 
 * the fake line endings.
 * 
 * @param[in] h_data Pointer to the csv data in host memory
 * @param[in] h_size Size of the input data, in bytes
 * @param[in,out] csvData Structure containing the csv parsing parameters
 * and intermediate results
 * 
 * @return gdf_error with error code on failure, otherwise GDF_SUCCESS
 *---------------------------------------------------------------------------**/
gdf_error launch_storeRecordStart(const char *h_data, size_t h_size,
                                  raw_csv_t *csvData) {

	char* d_chunk = nullptr;
	// Allocate extra byte in case \r\n is at the chunk border
	RMM_TRY(RMM_ALLOC (&d_chunk, max_chunk_bytes + 1, 0)); 
	
	cu_reccnt_t*	d_num_records;
	RMM_TRY(RMM_ALLOC((void**)&d_num_records, sizeof(cu_reccnt_t), 0) );
	CUDA_TRY(hipMemset(d_num_records, 0ull, sizeof(cu_reccnt_t)));

	int blockSize;		// suggested thread count to use
	int minGridSize;	// minimum block count required
	CUDA_TRY(hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, storeRecordStart) );

	const size_t chunk_count = (h_size + max_chunk_bytes - 1) / max_chunk_bytes;
	for (size_t ci = 0; ci < chunk_count; ++ci) {	
		const auto chunk_offset = ci * max_chunk_bytes;	
		const auto h_chunk = h_data + chunk_offset;
		const auto chunk_bytes = std::min((size_t)(h_size - ci * max_chunk_bytes), max_chunk_bytes);
		const auto chunk_bits = (chunk_bytes + 63) / 64;
		// include_first_row should only apply to the first chunk
		const bool cu_include_first_row = (ci == 0) && (csvData->byte_range_offset == 0);
		
		// Copy chunk to device. Copy extra byte if not last chunk
		CUDA_TRY(hipMemcpy(d_chunk, h_chunk, ci < (chunk_count - 1)?chunk_bytes:chunk_bytes + 1, hipMemcpyDefault));

		const int gridSize = (chunk_bits + blockSize - 1) / blockSize;
		storeRecordStart <<< gridSize, blockSize >>> (
			d_chunk, chunk_offset, csvData->opts.terminator, csvData->opts.quotechar, cu_include_first_row,
			chunk_bytes, chunk_bits, d_num_records,
			csvData->recStart
		);
	}

	RMM_TRY( RMM_FREE( d_num_records, 0 ) ); 
	RMM_TRY( RMM_FREE( d_chunk, 0 ) );

	CUDA_TRY( hipGetLastError() );

	return GDF_SUCCESS;
}


/**---------------------------------------------------------------------------*
 * @brief CUDA kernel that finds the start of each row (record) in the given 
 * file segment, based on the location of line terminators. 
 * 
 * The offset of each found row is stored in a device memory array. 
 * The kernel operate on a segment (chunk) of the csv file.
 * 
 * @param[in] data Device memory pointer to the csv data, 
 * potentially a chunk of the whole file
 * @param[in] chunk_offset Offset of the data pointer from the start of the file
 * @param[in] terminator Line terminator character
 * @param[in] quotechar Quote character
 * @param[in] num_bytes Number of bytes in the input data
 * @param[in] num_bits Number of 'bits' in the input data. Each 'bit' is
 * processed by a separate CUDA thread
 * @param[in,out] num_records Device memory pointer to the number of found rows
 * @param[out] recStart device memory array containing the offset of each record
 * 
 * @return void
 *---------------------------------------------------------------------------**/
__global__ void storeRecordStart(char *data, size_t chunk_offset, 
	const char terminator, const char quotechar, bool include_first_row,
	long num_bytes, long num_bits, cu_reccnt_t* num_records,
	cu_recstart_t* recStart) {

	// thread IDs range per block, so also need the block id
	const long tid = threadIdx.x + (blockDim.x * blockIdx.x);

	if ( tid >= num_bits)
		return;

	// data ID - multiple of 64
	const long did = tid * 64L;

	if (did == 0 && include_first_row) {
		const auto pos = atomicAdd(num_records, 1ull);
		recStart[pos] = 0;
	}

	const char *raw = (data + did);

	const long byteToProcess = ((did + 64L) < num_bytes) ? 64L : (num_bytes - did);

	// process the data
	for (long x = 0; x < byteToProcess; x++) {

		// Scan and log records. If quotations are enabled, then also log quotes
		// for a postprocess ignore, as the chunk here has limited visibility.
		if ((raw[x] == terminator) || (quotechar != '\0' && raw[x] == quotechar)) {

			const auto pos = atomicAdd(num_records, 1ull);
			recStart[pos] = did + chunk_offset + x + 1;

		} else if (terminator == '\n' && (x + 1L) < byteToProcess && 
				   raw[x] == '\r' && raw[x + 1L] == '\n') {

			x++;
			const auto pos = atomicAdd(num_records, 1ull);
			recStart[pos] = did + chunk_offset + x + 1;
		}

	}
}


/**---------------------------------------------------------------------------*
 * @brief Helper function to setup and launch CSV parsing CUDA kernel.
 * 
 * @param[in,out] raw_csv The metadata for the CSV data
 * @param[out] gdf The output column data
 * @param[out] valid The bitmaps indicating whether column fields are valid
 * @param[out] str_cols The start/end offsets for string data types
 * @param[out] num_valid The numbers of valid fields in columns
 *
 * @return gdf_error GDF_SUCCESS upon completion
 *---------------------------------------------------------------------------**/
gdf_error launch_dataConvertColumns(raw_csv_t *raw_csv, void **gdf,
                                    gdf_valid_type **valid, gdf_dtype *d_dtypes,
                                    string_pair **str_cols,
                                    unsigned long long *num_valid) {
  int blockSize;    // suggested thread count to use
  int minGridSize;  // minimum block count required
  CUDA_TRY(hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize,
                                              convertCsvToGdf));

  // Calculate actual block count to use based on records count
  int gridSize = (raw_csv->num_records + blockSize - 1) / blockSize;

  convertCsvToGdf <<< gridSize, blockSize >>> (
      raw_csv->data, raw_csv->opts, raw_csv->num_records,
      raw_csv->num_actual_cols, raw_csv->d_parseCol, raw_csv->recStart,
      d_dtypes,
      raw_csv->d_naTrie.empty() ? nullptr : raw_csv->d_naTrie.data().get(), gdf,
      valid, str_cols, num_valid);

  CUDA_TRY(hipGetLastError());
  return GDF_SUCCESS;
}

/**---------------------------------------------------------------------------*
 * @brief Functor for converting CSV data to cuDF data type value.
 *---------------------------------------------------------------------------**/
struct ConvertFunctor {
  /**---------------------------------------------------------------------------*
   * @brief Template specialization for operator() that handles integer types
   * that additionally checks whether the parsed data value should be overridden
   * with user-specified true/false matches.
   *
   * It is handled here rather than within convertStrToValue() as that function
   * is already used to construct the true/false match list from user-provided
   * strings at the start of parsing.
   *---------------------------------------------------------------------------**/
  template <typename T,
            typename std::enable_if_t<std::is_integral<T>::value> * = nullptr>
  __host__ __device__ __forceinline__ void operator()(
      const char *csvData, void *gdfColumnData, long rowIndex, long start,
      long end, const ParseOptions &opts) {
    T &value{static_cast<T *>(gdfColumnData)[rowIndex]};
    value = convertStrToValue<T>(csvData, start, end, opts);

    // Check for user-specified true/false values where the output is
    // replaced with 1/0 respectively
    if (isBooleanValue(value, opts.trueValues, opts.trueValuesCount)) {
      value = 1;
    } else if (isBooleanValue(value, opts.falseValues, opts.falseValuesCount)) {
      value = 0;
    }
  }

  /**---------------------------------------------------------------------------*
   * @brief Default template operator() dispatch specialization all data types
   * (including wrapper types) that is not covered by integral specialization.
   *---------------------------------------------------------------------------**/
  template <typename T,
            typename std::enable_if_t<!std::is_integral<T>::value> * = nullptr>
  __host__ __device__ __forceinline__ void operator()(
      const char *csvData, void *gdfColumnData, long rowIndex, long start,
      long end, const ParseOptions &opts) {
    T &value{static_cast<T *>(gdfColumnData)[rowIndex]};
    value = convertStrToValue<T>(csvData, start, end, opts);
  }
};

/**---------------------------------------------------------------------------*
 * @brief CUDA kernel iterates over the data until the end of the current field
 * 
 * Also iterates over (one or more) delimiter characters after the field.
 *
 * @param[in] raw_csv The entire CSV data to read
 * @param[in] opts A set of parsing options
 * @param[in] pos Offset to start the seeking from 
 * @param[in] stop Offset of the end of the row
 *
 * @return long position of the last character in the field, including the 
 *  delimiter(s) folloing the field data
 *---------------------------------------------------------------------------**/
__device__ 
long seekFieldEnd(const char *raw_csv, const ParseOptions opts, long pos, long stop) {
	bool quotation	= false;
	while(true){
		// Use simple logic to ignore control chars between any quote seq
		// Handles nominal cases including doublequotes within quotes, but
		// may not output exact failures as PANDAS for malformed fields
		if(raw_csv[pos] == opts.quotechar){
			quotation = !quotation;
		}
		else if(quotation==false){
			if(raw_csv[pos] == opts.delimiter){
				while (opts.multi_delimiter &&
					   pos < stop &&
					   raw_csv[pos + 1] == opts.delimiter) {
					++pos;
				}
				break;
			}
			else if(raw_csv[pos] == opts.terminator){
				break;
			}
			else if(raw_csv[pos] == '\r' && ((pos+1) < stop && raw_csv[pos+1] == '\n')){
				stop--;
				break;
			}
		}
		if(pos>=stop)
			break;
		pos++;
	}
	return pos;
}

/**---------------------------------------------------------------------------*
 * @brief CUDA kernel that parses and converts CSV data into cuDF column data.
 * 
 * Data is processed one record at a time
 *
 * @param[in] raw_csv The entire CSV data to read
 * @param[in] opts A set of parsing options
 * @param[in] num_records The number of lines/rows of CSV data
 * @param[in] num_columns The number of columns of CSV data
 * @param[in] parseCol Whether to parse or skip a column
 * @param[in] recStart The start the CSV data of interest
 * @param[in] dtype The data type of the column
 * @param[out] gdf_data The output column data
 * @param[out] valid The bitmaps indicating whether column fields are valid
 * @param[out] str_cols The start/end offsets for string data types
 * @param[out] num_valid The numbers of valid fields in columns
 *
 * @return gdf_error GDF_SUCCESS upon completion
 *---------------------------------------------------------------------------**/
__global__
void convertCsvToGdf(char *raw_csv,
                     const ParseOptions opts,
                     gdf_size_type num_records,
                     int num_columns,
                     bool *parseCol,
                     cu_recstart_t *recStart,
                     gdf_dtype *dtype,
                     SerialTrieNode* na_trie,
                     void **gdf_data,
                     gdf_valid_type **valid,
                     string_pair **str_cols,
                     unsigned long long *num_valid)
{
	// thread IDs range per block, so also need the block id
	long	rec_id  = threadIdx.x + (blockDim.x * blockIdx.x);		// this is entry into the field array - tid is an elements within the num_entries array

	// we can have more threads than data, make sure we are not past the end of the data
	if ( rec_id >= num_records)
		return;

	long start 		= recStart[rec_id];
	long stop 		= recStart[rec_id + 1];

	long pos 		= start;
	int  col 		= 0;
	int  actual_col = 0;
	int  stringCol 	= 0;

	while(col<num_columns){

		if(start>stop)
			break;

		pos = seekFieldEnd(raw_csv, opts, pos, stop);

		if(parseCol[col]==true){

			// check if the entire field is a NaN string - consistent with pandas
			const bool is_na = (na_trie == nullptr) ? false : serializedTrieContains(na_trie, raw_csv + start, pos - start);

			// Modify start & end to ignore whitespace and quotechars
			long tempPos=pos-1;
			if(!is_na && dtype[actual_col] != gdf_dtype::GDF_CATEGORY && dtype[actual_col] != gdf_dtype::GDF_STRING){
				adjustForWhitespaceAndQuotes(raw_csv, &start, &tempPos, opts.quotechar);
			}

			if(!is_na && start<=(tempPos)) { // Empty fields are not legal values

				// Type dispatcher does not handle GDF_STRINGS
				if (dtype[actual_col] == gdf_dtype::GDF_STRING) {
					long end = pos;
					if(opts.keepquotes==false){
						if((raw_csv[start] == opts.quotechar) && (raw_csv[end-1] == opts.quotechar)){
							start++;
							end--;
						}
					}
					str_cols[stringCol][rec_id].first	= raw_csv+start;
					str_cols[stringCol][rec_id].second	= size_t(end-start);
					stringCol++;
				} else {
					cudf::type_dispatcher(
						dtype[actual_col], ConvertFunctor{}, raw_csv,
						gdf_data[actual_col], rec_id, start, tempPos, opts);
				}

				// set the valid bitmap - all bits were set to 0 to start
				int bitmapIdx 	= whichBitmap(rec_id);  	// which bitmap
				int bitIdx		= whichBit(rec_id);		// which bit - over an 8-bit index
				setBit(valid[actual_col]+bitmapIdx, bitIdx);		// This is done with atomics

				atomicAdd((unsigned long long int*)&num_valid[actual_col],(unsigned long long int)1);
			}
			else if(dtype[actual_col]==gdf_dtype::GDF_STRING){
				str_cols[stringCol][rec_id].first 	= NULL;
				str_cols[stringCol][rec_id].second 	= 0;
				stringCol++;
			}
			actual_col++;
		}
		pos++;
		start=pos;
		col++;

	}
}

/**---------------------------------------------------------------------------*
 * @brief Helper function to setup and launch CSV data type detect CUDA kernel.
 * 
 * @param[in] raw_csv The metadata for the CSV data
 * @param[out] d_columnData The count for each column data type
 *
 * @return gdf_error GDF_SUCCESS upon completion
 *---------------------------------------------------------------------------**/
gdf_error launch_dataTypeDetection(raw_csv_t *raw_csv,
                                   column_data_t *d_columnData) {
  int blockSize;    // suggested thread count to use
  int minGridSize;  // minimum block count required
  CUDA_TRY(hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize,
                                              dataTypeDetection));

  // Calculate actual block count to use based on records count
  int gridSize = (raw_csv->num_records + blockSize - 1) / blockSize;

  dataTypeDetection <<< gridSize, blockSize >>> (
      raw_csv->data, raw_csv->opts, raw_csv->num_records,
      raw_csv->num_actual_cols, raw_csv->d_parseCol, raw_csv->recStart,
      d_columnData);

  CUDA_TRY(hipGetLastError());
  return GDF_SUCCESS;
}

/**---------------------------------------------------------------------------*
 * @brief CUDA kernel that parses and converts CSV data into cuDF column data.
 *
 * Data is processed in one row/record at a time, so the number of total
 * threads (tid) is equal to the number of rows.
 *
 * @Param[in] raw_csv The entire CSV data to read
 * @Param[in] opts A set of parsing options
 * @Param[in] num_records The number of lines/rows of CSV data
 * @Param[in] num_columns The number of columns of CSV data
 * @Param[in] parseCol Whether to parse or skip a column
 * @Param[in] recStart The start the CSV data of interest
 * @Param[out] d_columnData The count for each column data type
 *
 * @Returns GDF_SUCCESS upon successful computation
 *---------------------------------------------------------------------------**/
__global__
void dataTypeDetection(char *raw_csv,
                       const ParseOptions opts,
                       gdf_size_type num_records,
                       int num_columns,
                       bool *parseCol,
                       cu_recstart_t *recStart,
                       column_data_t *d_columnData)
{
	// thread IDs range per block, so also need the block id
	long	rec_id  = threadIdx.x + (blockDim.x * blockIdx.x);		// this is entry into the field array - tid is an elements within the num_entries array

	// we can have more threads than data, make sure we are not past the end of the data
	if ( rec_id >= num_records)
		return;

	long start 		= recStart[rec_id];
	long stop 		= recStart[rec_id + 1];

	long pos 		= start;
	int  col 		= 0;
	int  actual_col = 0;

	// Going through all the columns of a given record
	while(col<num_columns){

		if(start>stop)
			break;

		pos = seekFieldEnd(raw_csv, opts, pos, stop);

		// Checking if this is a column that the user wants --- user can filter columns
		if(parseCol[col]==true){

			long tempPos=pos-1;

			// Checking if the record is NULL
			if(start>(tempPos)){
				atomicAdd(& d_columnData[actual_col].countNULL, 1L);
				pos++;
				start=pos;
				col++;
				actual_col++;
				continue;	
			}

			long countNumber=0;
			long countDecimal=0;
			long countSlash=0;
			long countDash=0;
			long countColon=0;
			long countString=0;

			// Modify start & end to ignore whitespace and quotechars
			// This could possibly result in additional empty fields
			adjustForWhitespaceAndQuotes(raw_csv, &start, &tempPos);

			long strLen=tempPos-start+1;

			for(long startPos=start; startPos<=tempPos; startPos++){
				if(raw_csv[startPos]>= '0' && raw_csv[startPos] <= '9'){
					countNumber++;
					continue;
				}
				// Looking for unique characters that will help identify column types.
				switch (raw_csv[startPos]){
					case '.':
						countDecimal++;break;
					case '-':
						countDash++; break;
					case '/':
						countSlash++;break;
					case ':':
						countColon++;break;
					default:
						countString++;
						break;	
				}
			}

			if(strLen==0){ // Removed spaces ' ' in the pre-processing and thus we can have an empty string.
				atomicAdd(& d_columnData[actual_col].countNULL, 1L);
			}
			// Integers have to have the length of the string or can be off by one if they start with a minus sign
			else if(countNumber==(strLen) || ( strLen>1 && countNumber==(strLen-1) && raw_csv[start]=='-') ){
				// Checking to see if we the integer value requires 8,16,32,64 bits.
				// This will allow us to allocate the exact amount of memory.
				const auto value = convertStrToValue<int64_t>(raw_csv, start, tempPos, opts);

				if (isBooleanValue<int32_t>(value, opts.trueValues, opts.trueValuesCount) ||
					isBooleanValue<int32_t>(value, opts.falseValues, opts.falseValuesCount)){
					atomicAdd(& d_columnData[actual_col].countInt8, 1L);
				}
				else if(value >= (1L<<31)){
					atomicAdd(& d_columnData[actual_col].countInt64, 1L);
				}
				else if(value >= (1L<<15)){
					atomicAdd(& d_columnData[actual_col].countInt32, 1L);
				}
				else if(value >= (1L<<7)){
					atomicAdd(& d_columnData[actual_col].countInt16, 1L);
				}
				else{
					atomicAdd(& d_columnData[actual_col].countInt8, 1L);
				}
			}
			// Floating point numbers are made up of numerical strings, have to have a decimal sign, and can have a minus sign.
			else if((countNumber==(strLen-1) && countDecimal==1) || (strLen>2 && countNumber==(strLen-2) && raw_csv[start]=='-')){
					atomicAdd(& d_columnData[actual_col].countFloat, 1L);
			}
			// The date-time field cannot have more than 3 strings. As such if an entry has more than 3 string characters, it is not 
			// a data-time field. Also, if a string has multiple decimals, then is not a legit number.
			else if(countString > 3 || countDecimal > 1){
				atomicAdd(& d_columnData[actual_col].countString, 1L);
			}
			else {
				// A date field can have either one or two '-' or '\'. A legal combination will only have one of them.
				// To simplify the process of auto column detection, we are not covering all the date-time formation permutations.
				if((countDash>0 && countDash<=2 && countSlash==0)|| (countDash==0 && countSlash>0 && 	countSlash<=2) ){
					if((countColon<=2)){
						atomicAdd(& d_columnData[actual_col].countDateAndTime, 1L);
					}
					else{
						atomicAdd(& d_columnData[actual_col].countString, 1L);					
					}
				}
				// Default field is string type.
				else{
					atomicAdd(& d_columnData[actual_col].countString, 1L);					
				}
			}
			actual_col++;
		}
		pos++;
		start=pos;
		col++;	

	}
}
