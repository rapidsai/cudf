/*
 * Copyright (c) 2020-2021, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

/**
 * @file writer_impl.cu
 * @brief cuDF-IO CSV writer class implementation
 */

#include "writer_impl.hpp"

#include <cudf/column/column_device_view.cuh>
#include <cudf/copying.hpp>
#include <cudf/detail/null_mask.hpp>
#include <cudf/null_mask.hpp>
#include <cudf/scalar/scalar.hpp>
#include <cudf/strings/detail/combine.hpp>
#include <cudf/strings/detail/converters.hpp>
#include <cudf/strings/detail/replace.hpp>
#include <cudf/strings/detail/utilities.cuh>

#include <rmm/cuda_stream_view.hpp>
#include <rmm/exec_policy.hpp>
#include <rmm/mr/device/per_device_resource.hpp>

#include <thrust/execution_policy.h>
#include <thrust/logical.h>
#include <thrust/scan.h>

#include <algorithm>
#include <sstream>

namespace cudf {
namespace io {
namespace detail {
namespace csv {

namespace {

/**
 * @brief Functor to modify a string column for CSV format.
 *
 * If a row contains specific characters, the entire row must be
 * output in double-quotes. Also, if a double-quote appears it
 * must be escaped using a 2nd double-quote.
 */
struct escape_strings_fn {
  column_device_view const d_column;
  string_view const d_delimiter;  // check for column delimiter
  offset_type* d_offsets{};
  char* d_chars{};

  __device__ void write_char(char_utf8 chr, char*& d_buffer, offset_type& bytes)
  {
    if (d_buffer)
      d_buffer += cudf::strings::detail::from_char_utf8(chr, d_buffer);
    else
      bytes += cudf::strings::detail::bytes_in_char_utf8(chr);
  }

  __device__ void operator()(size_type idx)
  {
    if (d_column.is_null(idx)) {
      if (!d_chars) d_offsets[idx] = 0;
      return;
    }

    constexpr char_utf8 const quote    = '\"';  // check for quote
    constexpr char_utf8 const new_line = '\n';  // and for new-line

    auto const d_str = d_column.element<string_view>(idx);

    // if quote, new-line or a column delimiter appear in the string
    // the entire string must be double-quoted.
    bool const quote_row = thrust::any_of(
      thrust::seq, d_str.begin(), d_str.end(), [d_delimiter = d_delimiter](auto chr) {
        return chr == quote || chr == new_line || chr == d_delimiter[0];
      });

    char* d_buffer    = d_chars ? d_chars + d_offsets[idx] : nullptr;
    offset_type bytes = 0;

    if (quote_row) write_char(quote, d_buffer, bytes);
    for (auto chr : d_str) {
      if (chr == quote) write_char(quote, d_buffer, bytes);
      write_char(chr, d_buffer, bytes);
    }
    if (quote_row) write_char(quote, d_buffer, bytes);

    if (!d_chars) d_offsets[idx] = bytes;
  }
};

struct column_to_strings_fn {
  // compile-time predicate that defines unsupported column types;
  // based on the conditions used for instantiations of individual
  // converters in strings/convert/convert_*.hpp;
  //(this should have been a `variable template`,
  // instead of a static function, but nvcc (10.0)
  // fails to compile var-templs);
  //
  template <typename column_type>
  constexpr static bool is_not_handled(void)
  {
    // Note: the case (not std::is_same<column_type, bool>::value)
    // is already covered by is_integral)
    //
    return not((std::is_same<column_type, cudf::string_view>::value) ||
               (std::is_integral<column_type>::value) ||
               (std::is_floating_point<column_type>::value) ||
               (cudf::is_fixed_point<column_type>()) || (cudf::is_timestamp<column_type>()) ||
               (cudf::is_duration<column_type>()));
  }

  explicit column_to_strings_fn(
    csv_writer_options const& options,
    rmm::cuda_stream_view stream        = rmm::cuda_stream_default,
    rmm::mr::device_memory_resource* mr = rmm::mr::get_current_device_resource())
    : options_(options), stream_(stream), mr_(mr)
  {
  }

  // Note: `null` replacement with `na_rep` deferred to `concatenate()`
  // instead of column-wise; might be faster
  //
  // Note: Cannot pass `stream` to detail::<fname> version of <fname> calls below, because they are
  // not exposed in header (see, for example, detail::concatenate(tbl_view, separator, na_rep, mr,
  // stream) is declared and defined in combine.cu); Possible solution: declare `extern`, or just
  // declare a prototype inside `namespace cudf::strings::detail`;

  // bools:
  //
  template <typename column_type>
  std::enable_if_t<std::is_same<column_type, bool>::value, std::unique_ptr<column>> operator()(
    column_view const& column) const
  {
    return cudf::strings::detail::from_booleans(
      column, options_.get_true_value(), options_.get_false_value(), stream_, mr_);
  }

  // strings:
  //
  template <typename column_type>
  std::enable_if_t<std::is_same<column_type, cudf::string_view>::value, std::unique_ptr<column>>
  operator()(column_view const& column_v) const
  {
    // handle special characters: {delimiter, '\n', "} in row:
    string_scalar delimiter{std::string{options_.get_inter_column_delimiter()}, true, stream_};

    auto d_column = column_device_view::create(column_v, stream_);
    escape_strings_fn fn{*d_column, delimiter.value(stream_)};
    auto children = cudf::strings::detail::make_strings_children(fn, column_v.size(), stream_, mr_);

    return make_strings_column(column_v.size(),
                               std::move(children.first),
                               std::move(children.second),
                               column_v.null_count(),
                               cudf::detail::copy_bitmask(column_v, stream_, mr_),
                               stream_,
                               mr_);
  }

  // ints:
  //
  template <typename column_type>
  std::enable_if_t<std::is_integral<column_type>::value && !std::is_same<column_type, bool>::value,
                   std::unique_ptr<column>>
  operator()(column_view const& column) const
  {
    return cudf::strings::detail::from_integers(column, stream_, mr_);
  }

  // floats:
  //
  template <typename column_type>
  std::enable_if_t<std::is_floating_point<column_type>::value, std::unique_ptr<column>> operator()(
    column_view const& column) const
  {
    return cudf::strings::detail::from_floats(column, stream_, mr_);
  }

  // fixed point:
  //
  template <typename column_type>
  std::enable_if_t<cudf::is_fixed_point<column_type>(), std::unique_ptr<column>> operator()(
    column_view const& column) const
  {
    return cudf::strings::detail::from_fixed_point(column, stream_, mr_);
  }

  // timestamps:
  //
  template <typename column_type>
  std::enable_if_t<cudf::is_timestamp<column_type>(), std::unique_ptr<column>> operator()(
    column_view const& column) const
  {
    std::string format = [&]() {
      if (std::is_same<cudf::timestamp_s, column_type>::value) {
        return std::string{"%Y-%m-%dT%H:%M:%SZ"};
      } else if (std::is_same<cudf::timestamp_ms, column_type>::value) {
        return std::string{"%Y-%m-%dT%H:%M:%S.%3fZ"};
      } else if (std::is_same<cudf::timestamp_us, column_type>::value) {
        return std::string{"%Y-%m-%dT%H:%M:%S.%6fZ"};
      } else if (std::is_same<cudf::timestamp_ns, column_type>::value) {
        return std::string{"%Y-%m-%dT%H:%M:%S.%9fZ"};
      } else {
        return std::string{"%Y-%m-%d"};
      }
    }();

    // handle the cases where delimiter / line-terminator can be
    // "-" or ":", in which case we need to add quotes to the format
    //
    std::string delimiter{options_.get_inter_column_delimiter()};
    std::string newline{options_.get_line_terminator()};

    constexpr char const* dash{"-"};
    constexpr char const* colon{":"};
    if (delimiter == dash || newline == dash || delimiter == colon || newline == colon) {
      format = "\"" + format + "\"";
    }

    return cudf::strings::detail::from_timestamps(column, format, stream_, mr_);
  }

  template <typename column_type>
  std::enable_if_t<cudf::is_duration<column_type>(), std::unique_ptr<column>> operator()(
    column_view const& column) const
  {
    return cudf::io::detail::csv::pandas_format_durations(column, stream_, mr_);
  }

  // unsupported type of column:
  //
  template <typename column_type>
  std::enable_if_t<is_not_handled<column_type>(), std::unique_ptr<column>> operator()(
    column_view const&) const
  {
    CUDF_FAIL("Unsupported column type.");
  }

 private:
  csv_writer_options const& options_;
  rmm::cuda_stream_view stream_;
  rmm::mr::device_memory_resource* mr_;
};
}  // unnamed namespace

// Forward to implementation
writer::writer(std::unique_ptr<data_sink> sink,
               csv_writer_options const& options,
               rmm::cuda_stream_view stream,
               rmm::mr::device_memory_resource* mr)
  : _impl(std::make_unique<impl>(std::move(sink), options, mr))
{
}

// Destructor within this translation unit
writer::~writer() = default;

writer::impl::impl(std::unique_ptr<data_sink> sink,
                   csv_writer_options const& options,
                   rmm::mr::device_memory_resource* mr)
  : out_sink_(std::move(sink)), mr_(mr), options_(options)
{
}

// write the header: column names:
//
void writer::impl::write_chunked_begin(table_view const& table,
                                       const table_metadata* metadata,
                                       rmm::cuda_stream_view stream)
{
  if ((metadata != nullptr) && (options_.is_enabled_include_header())) {
    CUDF_EXPECTS(metadata->column_names.size() == static_cast<size_t>(table.num_columns()),
                 "Mismatch between number of column headers and table columns.");
    std::string delimiter_str{options_.get_inter_column_delimiter()};

    // avoid delimiter after last element:
    //
    std::stringstream ss;
    std::copy(metadata->column_names.begin(),
              metadata->column_names.end() - 1,
              std::ostream_iterator<std::string>(ss, delimiter_str.c_str()));

    if (metadata->column_names.size() > 0) {
      ss << metadata->column_names.back() << options_.get_line_terminator();
    } else {
      ss << options_.get_line_terminator();
    }

    out_sink_->host_write(ss.str().data(), ss.str().size());
  }
}

void writer::impl::write_chunked(strings_column_view const& str_column_view,
                                 const table_metadata* metadata,
                                 rmm::cuda_stream_view stream)
{
  // algorithm outline:
  //
  //  for_each(strings_column.begin(), strings_column.end(),
  //           [sink = out_sink_](auto str_row) mutable {
  //               auto host_buffer = str_row.host_buffer();
  //               sink->host_write(host_buffer_.data(), host_buffer_.size());
  //           });//or...sink->device_write(device_buffer,...);
  //
  // added line_terminator functionality
  //

  CUDF_EXPECTS(str_column_view.size() > 0, "Unexpected empty strings column.");

  cudf::string_scalar newline{options_.get_line_terminator()};
  auto p_str_col_w_nl =
    cudf::strings::detail::join_strings(str_column_view, newline, string_scalar("", false), stream);
  strings_column_view strings_column{p_str_col_w_nl->view()};

  auto total_num_bytes      = strings_column.chars_size();
  char const* ptr_all_bytes = strings_column.chars().data<char>();

  if (out_sink_->is_device_write_preferred(total_num_bytes)) {
    // Direct write from device memory
    out_sink_->device_write(ptr_all_bytes, total_num_bytes, stream);
  } else {
    // copy the bytes to host to write them out
    thrust::host_vector<char> h_bytes(total_num_bytes);
    CUDA_TRY(hipMemcpyAsync(h_bytes.data(),
                             ptr_all_bytes,
                             total_num_bytes * sizeof(char),
                             hipMemcpyDeviceToHost,
                             stream.value()));
    stream.synchronize();

    out_sink_->host_write(h_bytes.data(), total_num_bytes);
  }

  // Needs newline at the end, to separate from next chunk
  if (out_sink_->is_device_write_preferred(newline.size())) {
    out_sink_->device_write(newline.data(), newline.size(), stream);
  } else {
    out_sink_->host_write(options_.get_line_terminator().data(),
                          options_.get_line_terminator().size());
  }
}

void writer::impl::write(table_view const& table,
                         const table_metadata* metadata,
                         rmm::cuda_stream_view stream)
{
  // write header: column names separated by delimiter:
  // (even for tables with no rows)
  //
  write_chunked_begin(table, metadata, stream);

  if (table.num_rows() > 0) {
    // no need to check same-size columns constraint; auto-enforced by table_view
    auto n_rows_per_chunk = options_.get_rows_per_chunk();
    //
    // This outputs the CSV in row chunks to save memory.
    // Maybe we can use the total_rows*count calculation and a memory threshold
    // instead of an arbitrary chunk count.
    // The entire CSV chunk must fit in CPU memory before writing it out.
    //
    if (n_rows_per_chunk % 8)  // must be divisible by 8
      n_rows_per_chunk += 8 - (n_rows_per_chunk % 8);

    CUDF_EXPECTS(n_rows_per_chunk >= 8, "write_csv: invalid chunk_rows; must be at least 8");

    auto num_rows = table.num_rows();
    std::vector<table_view> vector_views;

    if (num_rows <= n_rows_per_chunk) {
      vector_views.push_back(table);
    } else {
      auto const n_chunks = num_rows / n_rows_per_chunk;
      std::vector<size_type> splits(n_chunks);
      thrust::tabulate(splits.begin(), splits.end(), [n_rows_per_chunk](auto idx) {
        return (idx + 1) * n_rows_per_chunk;
      });

      // split table_view into chunks:
      vector_views = cudf::split(table, splits);
    }

    // convert each chunk to CSV:
    //
    column_to_strings_fn converter{options_, stream, rmm::mr::get_current_device_resource()};
    for (auto&& sub_view : vector_views) {
      // Skip if the table has no rows
      if (sub_view.num_rows() == 0) continue;
      std::vector<std::unique_ptr<column>> str_column_vec;

      // populate vector of string-converted columns:
      //
      std::transform(sub_view.begin(),
                     sub_view.end(),
                     std::back_inserter(str_column_vec),
                     [converter](auto const& current_col) {
                       return cudf::type_dispatcher(current_col.type(), converter, current_col);
                     });

      // create string table view from str_column_vec:
      //
      auto str_table_ptr  = std::make_unique<cudf::table>(std::move(str_column_vec));
      auto str_table_view = str_table_ptr->view();

      // concatenate columns in each row into one big string column
      // (using null representation and delimiter):
      //
      std::string delimiter_str{options_.get_inter_column_delimiter()};
      auto str_concat_col = [&] {
        if (str_table_view.num_columns() > 1)
          return cudf::strings::detail::concatenate(str_table_view,
                                                    delimiter_str,
                                                    options_.get_na_rep(),
                                                    strings::separator_on_nulls::YES,
                                                    stream);
        cudf::string_scalar narep{options_.get_na_rep()};
        return cudf::strings::detail::replace_nulls(str_table_view.column(0), narep, stream);
      }();

      write_chunked(str_concat_col->view(), metadata, stream);
    }
  }

  // finalize (no-op, for now, but offers a hook for future extensions):
  //
  write_chunked_end(table, metadata, stream);
}

void writer::write(table_view const& table,
                   const table_metadata* metadata,
                   rmm::cuda_stream_view stream)
{
  _impl->write(table, metadata, stream);
}

}  // namespace csv
}  // namespace detail
}  // namespace io
}  // namespace cudf
