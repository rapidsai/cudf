/*
 * Copyright (c) 2020, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

/**
 * @file writer_impl.cu
 * @brief cuDF-IO CSV writer class implementation
 */

#include "writer_impl.hpp"

#include <strings/utilities.cuh>

#include <cudf/copying.hpp>
#include <cudf/null_mask.hpp>
#include <cudf/scalar/scalar.hpp>
#include <cudf/strings/combine.hpp>
#include <cudf/strings/convert/convert_booleans.hpp>
#include <cudf/strings/convert/convert_datetime.hpp>
#include <cudf/strings/convert/convert_floats.hpp>
#include <cudf/strings/convert/convert_integers.hpp>
#include <cudf/strings/detail/modify_strings.cuh>
#include <cudf/strings/replace.hpp>
#include <cudf/utilities/traits.hpp>

#include <rmm/cuda_stream_view.hpp>
#include <rmm/exec_policy.hpp>

#include <thrust/count.h>
#include <thrust/execution_policy.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/iterator/discard_iterator.h>
#include <thrust/scan.h>
#include <thrust/transform.h>

#include <algorithm>
#include <cstring>
#include <iterator>
#include <sstream>
#include <type_traits>
#include <utility>

namespace cudf {
namespace io {
namespace detail {
namespace csv {

namespace {  // anonym.
// helpers:

using namespace cudf::strings;

// predicate to determine if a given string_view contains special characters:
//{"\"", "\n", <delimiter>}
//
struct predicate_special_chars {
  explicit predicate_special_chars(string_view const& delimiter) : delimiter_(delimiter) {}

  __device__ bool operator()(string_view const& str_view) const
  {
    // if (any_of{"\"", "\n", <delimiter>} )
    //
    constexpr char const* quote_str   = "\"";
    constexpr char const* newline_str = "\n";
    constexpr size_type len1byte{1};

    if ((str_view.find(quote_str, len1byte) >= 0) || (str_view.find(newline_str, len1byte) >= 0) ||
        (str_view.find(delimiter_) >= 0)) {
      return true;
    } else {
      return false;
    }
  }

 private:
  string_view const delimiter_;
};

struct probe_special_chars {
  probe_special_chars(column_device_view const d_column, predicate_special_chars const& predicate)
    : d_column_(d_column), predicate_(predicate)
  {
  }

  __device__ int32_t operator()(size_type idx) const
  {
    if (d_column_.is_null(idx)) {
      return 0;  // null string, so no-op
    }

    string_view d_str = d_column_.template element<string_view>(idx);

    if (predicate_(d_str)) {
      constexpr char const quote_char = '\"';

      // count number of quotes "\""
      size_type num_quotes = thrust::count_if(
        thrust::seq, d_str.begin(), d_str.end(), [](char_utf8 chr) { return chr == quote_char; });
      return d_str.size_bytes() + num_quotes + 2;
    } else {
      return d_str.size_bytes();
    }
  }

 private:
  column_device_view const d_column_;
  predicate_special_chars const predicate_;
};

struct modify_special_chars {
  modify_special_chars(column_device_view const d_column,
                       int32_t const* d_offsets,
                       char* d_chars,
                       predicate_special_chars const& predicate)
    : d_column_(d_column), d_offsets_(d_offsets), d_chars_(d_chars), predicate_(predicate)
  {
  }

  __device__ int32_t operator()(size_type idx)
  {
    using namespace cudf::strings::detail;

    if (d_column_.is_null(idx)) {
      return 0;  // null string, so no-op
    }

    string_view d_str        = d_column_.template element<string_view>(idx);
    size_type str_size_bytes = d_str.size_bytes();

    char* d_buffer = get_output_ptr(idx);
    // assert( d_buffer != nullptr );

    if (predicate_(d_str)) {
      constexpr char const quote_char   = '\"';
      constexpr char const* quote_str   = "\"";
      constexpr char const* str_2quotes = "\"\"";

      size_type len1quote{1};
      size_type len2quotes{2};

      // modify d_str by duplicating all 2bl quotes
      // and surrounding whole string by 2bl quotes:
      //
      // pre-condition: `d_str` is _not_ modified by `d_buffer` manipulation
      // because it's a copy of `idx` entry in `d_column_`
      //(since `d_column` is const)
      //
      d_buffer = copy_and_increment(d_buffer, quote_str, len1quote);  // add the quote prefix

      for (auto itr = d_str.begin(); itr != d_str.end(); ++itr) {
        char_utf8 the_chr = *itr;

        if (the_chr == quote_char) {
          d_buffer = copy_and_increment(d_buffer, str_2quotes, len2quotes);  // double the quote;
        } else {
          d_buffer += from_char_utf8(the_chr, d_buffer);
        }
      }

      d_buffer = copy_and_increment(d_buffer, quote_str, len1quote);  // add the quote suffix;
    } else {
      // copy the source string unmodified:
      //(pass-through)
      //
      memcpy(d_buffer, d_str.data(), str_size_bytes);
    }
    return 0;
  }

  __device__ char* get_output_ptr(size_type idx)
  {
    return d_chars_ && d_offsets_ ? d_chars_ + d_offsets_[idx] : nullptr;
  }

 private:
  column_device_view const d_column_;
  int32_t const* d_offsets_;
  char* d_chars_;
  predicate_special_chars const predicate_;
};

struct column_to_strings_fn {
  // compile-time predicate that defines unsupported column types;
  // based on the conditions used for instantiations of individual
  // converters in strings/convert/convert_*.hpp;
  //(this should have been a `variable template`,
  // instead of a static function, but nvcc (10.0)
  // fails to compile var-templs);
  //
  template <typename column_type>
  constexpr static bool is_not_handled(void)
  {
    // Note: the case (not std::is_same<column_type, bool>::value)
    // is already covered by is_integral)
    //
    return not((std::is_same<column_type, cudf::string_view>::value) ||
               (std::is_integral<column_type>::value) ||
               (std::is_floating_point<column_type>::value) ||
               (cudf::is_timestamp<column_type>()) || (cudf::is_duration<column_type>()));
  }

  explicit column_to_strings_fn(csv_writer_options const& options,
                                rmm::mr::device_memory_resource* mr = nullptr,
                                rmm::cuda_stream_view stream        = nullptr)
    : options_(options), mr_(mr), stream_(stream)
  {
  }

  // Note: `null` replacement with `na_rep` deferred to `concatenate()`
  // instead of column-wise; might be faster
  //
  // Note: Cannot pass `stream` to detail::<fname> version of <fname> calls below, because they are
  // not exposed in header (see, for example, detail::concatenate(tbl_view, separator, na_rep, mr,
  // stream) is declared and defined in combine.cu); Possible solution: declare `extern`, or just
  // declare a prototype inside `namespace cudf::strings::detail`;

  // bools:
  //
  template <typename column_type>
  std::enable_if_t<std::is_same<column_type, bool>::value, std::unique_ptr<column>> operator()(
    column_view const& column) const
  {
    auto conv_col_ptr = cudf::strings::from_booleans(
      column, options_.get_true_value(), options_.get_false_value(), mr_);

    return conv_col_ptr;
  }

  // strings:
  //
  template <typename column_type>
  std::enable_if_t<std::is_same<column_type, cudf::string_view>::value, std::unique_ptr<column>>
  operator()(column_view const& column_v) const
  {
    using namespace cudf::strings::detail;

    // handle special characters: {delimiter, '\n', "} in row:
    //
    // algorithm outline:
    //
    // target = "\"";
    // repl = ""\"\";
    //
    // str_column_ref = {};
    // for each str_row: column_v {
    //    if ((not null str_row) &&
    //        (str_row.find("\n") || str_row.find("\"") || str_row.find(delimiter) ))
    //        str_column_modified = modify(str_row);
    // where modify() = duplicate the double quotes, if any; add 2bl quotes prefix/suffix;
    //}
    //
    string_scalar delimiter{std::string{options_.get_inter_column_delimiter()}, true, stream_};
    predicate_special_chars pred{delimiter.value(stream_)};

    return modify_strings<probe_special_chars, modify_special_chars>(column_v, stream_, mr_, pred);
  }

  // ints:
  //
  template <typename column_type>
  std::enable_if_t<std::is_integral<column_type>::value && !std::is_same<column_type, bool>::value,
                   std::unique_ptr<column>>
  operator()(column_view const& column) const
  {
    auto conv_col_ptr = cudf::strings::from_integers(column, mr_);

    return conv_col_ptr;
  }

  // floats:
  //
  template <typename column_type>
  std::enable_if_t<std::is_floating_point<column_type>::value, std::unique_ptr<column>> operator()(
    column_view const& column) const
  {
    auto conv_col_ptr = cudf::strings::from_floats(column, mr_);

    return conv_col_ptr;
  }

  // timestamps:
  //
  template <typename column_type>
  std::enable_if_t<cudf::is_timestamp<column_type>(), std::unique_ptr<column>> operator()(
    column_view const& column) const
  {
    std::string format = [&]() {
      if (std::is_same<cudf::timestamp_s, column_type>::value) {
        return std::string{"%Y-%m-%dT%H:%M:%SZ"};
      } else if (std::is_same<cudf::timestamp_ms, column_type>::value) {
        return std::string{"%Y-%m-%dT%H:%M:%S.%3fZ"};
      } else if (std::is_same<cudf::timestamp_us, column_type>::value) {
        return std::string{"%Y-%m-%dT%H:%M:%S.%6fZ"};
      } else if (std::is_same<cudf::timestamp_ns, column_type>::value) {
        return std::string{"%Y-%m-%dT%H:%M:%S.%9fZ"};
      } else {
        return std::string{"%Y-%m-%d"};
      }
    }();

    // handle the cases where delimiter / line-terminator can be
    // "-" or ":", in which case they are to be dropped from the format:
    //
    std::string delimiter{options_.get_inter_column_delimiter()};
    std::string newline{options_.get_line_terminator()};

    constexpr char const* dash{"-"};
    constexpr char const* colon{":"};
    if (delimiter == dash || newline == dash) {
      format.erase(std::remove(format.begin(), format.end(), dash[0]), format.end());
    }

    if (delimiter == colon || newline == colon) {
      format.erase(std::remove(format.begin(), format.end(), colon[0]), format.end());
    }

    auto conv_col_ptr = cudf::strings::from_timestamps(column, format, mr_);

    return conv_col_ptr;
  }

  template <typename column_type>
  std::enable_if_t<cudf::is_duration<column_type>(), std::unique_ptr<column>> operator()(
    column_view const& column) const
  {
    return cudf::io::detail::csv::pandas_format_durations(column, stream_);
  }

  // unsupported type of column:
  //
  template <typename column_type>
  std::enable_if_t<is_not_handled<column_type>(), std::unique_ptr<column>> operator()(
    column_view const& column) const
  {
    CUDF_FAIL("Unsupported column type.");
  }

 private:
  csv_writer_options const& options_;
  rmm::mr::device_memory_resource* mr_;
  rmm::cuda_stream_view stream_;
};
}  // unnamed namespace

// Forward to implementation
writer::writer(std::unique_ptr<data_sink> sink,
               csv_writer_options const& options,
               rmm::mr::device_memory_resource* mr)
  : _impl(std::make_unique<impl>(std::move(sink), options, mr))
{
}

// Destructor within this translation unit
writer::~writer() = default;

writer::impl::impl(std::unique_ptr<data_sink> sink,
                   csv_writer_options const& options,
                   rmm::mr::device_memory_resource* mr)
  : out_sink_(std::move(sink)), mr_(mr), options_(options)
{
}

// write the header: column names:
//
void writer::impl::write_chunked_begin(table_view const& table,
                                       const table_metadata* metadata,
                                       rmm::cuda_stream_view stream)
{
  if ((metadata != nullptr) && (options_.is_enabled_include_header())) {
    CUDF_EXPECTS(metadata->column_names.size() == static_cast<size_t>(table.num_columns()),
                 "Mismatch between number of column headers and table columns.");

    std::string delimiter_str{options_.get_inter_column_delimiter()};

    // avoid delimiter after last element:
    //
    std::stringstream ss;
    std::copy(metadata->column_names.begin(),
              metadata->column_names.end() - 1,
              std::ostream_iterator<std::string>(ss, delimiter_str.c_str()));
    ss << metadata->column_names.back() << options_.get_line_terminator();

    out_sink_->host_write(ss.str().data(), ss.str().size());
  }
}

void writer::impl::write_chunked(strings_column_view const& str_column_view,
                                 const table_metadata* metadata,
                                 rmm::cuda_stream_view stream)
{
  // algorithm outline:
  //
  //  for_each(strings_column.begin(), strings_column.end(),
  //           [sink = out_sink_](auto str_row) mutable {
  //               auto host_buffer = str_row.host_buffer();
  //               sink->host_write(host_buffer_.data(), host_buffer_.size());
  //           });//or...sink->device_write(device_buffer,...);
  //
  // added line_terminator functionality
  //

  CUDF_EXPECTS(str_column_view.size() > 0, "Unexpected empty strings column.");

  cudf::string_scalar newline{options_.get_line_terminator()};
  auto p_str_col_w_nl = cudf::strings::join_strings(str_column_view, newline);
  strings_column_view strings_column{p_str_col_w_nl->view()};

  auto total_num_bytes      = strings_column.chars_size();
  char const* ptr_all_bytes = strings_column.chars().data<char>();

  if (out_sink_->supports_device_write()) {
    // host algorithm call, but the underlying call
    // is a device_write taking a device buffer;
    //
    out_sink_->device_write(ptr_all_bytes, total_num_bytes, stream);
    out_sink_->device_write(newline.data(),
                            newline.size(),
                            stream);  // needs newline at the end, to separate from next chunk
  } else {
    // no device write possible;
    //
    // copy the bytes to host, too:
    //
    thrust::host_vector<char> h_bytes(total_num_bytes);
    CUDA_TRY(hipMemcpyAsync(h_bytes.data(),
                             ptr_all_bytes,
                             total_num_bytes * sizeof(char),
                             hipMemcpyDeviceToHost,
                             stream.value()));

    stream.synchronize();

    // host algorithm call, where the underlying call
    // is also host_write taking a host buffer;
    //
    char const* ptr_h_bytes = h_bytes.data();
    out_sink_->host_write(ptr_h_bytes, total_num_bytes);
    out_sink_->host_write(options_.get_line_terminator().data(),
                          options_.get_line_terminator()
                            .size());  // needs newline at the end, to separate from next chunk
  }
}

void writer::impl::write(table_view const& table,
                         const table_metadata* metadata,
                         rmm::cuda_stream_view stream)
{
  CUDF_EXPECTS(table.num_columns() > 0, "Empty table.");

  // write header: column names separated by delimiter:
  // (even for tables with no rows)
  //
  write_chunked_begin(table, metadata, stream);

  if (table.num_rows() > 0) {
    // no need to check same-size columns constraint; auto-enforced by table_view
    auto n_rows_per_chunk = options_.get_rows_per_chunk();
    //
    // This outputs the CSV in row chunks to save memory.
    // Maybe we can use the total_rows*count calculation and a memory threshold
    // instead of an arbitrary chunk count.
    // The entire CSV chunk must fit in CPU memory before writing it out.
    //
    if (n_rows_per_chunk % 8)  // must be divisible by 8
      n_rows_per_chunk += 8 - (n_rows_per_chunk % 8);

    CUDF_EXPECTS(n_rows_per_chunk >= 8, "write_csv: invalid chunk_rows; must be at least 8");

    auto num_rows = table.num_rows();
    std::vector<table_view> vector_views;

    if (num_rows <= n_rows_per_chunk) {
      vector_views.push_back(table);
    } else {
      std::vector<size_type> splits;
      auto n_chunks = num_rows / n_rows_per_chunk;
      splits.resize(n_chunks);

      rmm::device_vector<size_type> d_splits(n_chunks, n_rows_per_chunk);
      thrust::inclusive_scan(
        rmm::exec_policy(stream), d_splits.begin(), d_splits.end(), d_splits.begin());

      CUDA_TRY(hipMemcpyAsync(splits.data(),
                               d_splits.data().get(),
                               n_chunks * sizeof(size_type),
                               hipMemcpyDeviceToHost,
                               stream.value()));

      stream.synchronize();

      // split table_view into chunks:
      //
      vector_views = cudf::split(table, splits);
    }

    // convert each chunk to CSV:
    //
    column_to_strings_fn converter{options_, mr_};
    for (auto&& sub_view : vector_views) {
      // Skip if the table has no rows
      if (sub_view.num_rows() == 0) continue;
      std::vector<std::unique_ptr<column>> str_column_vec;

      // populate vector of string-converted columns:
      //
      std::transform(sub_view.begin(),
                     sub_view.end(),
                     std::back_inserter(str_column_vec),
                     [converter](auto const& current_col) {
                       return cudf::type_dispatcher(current_col.type(), converter, current_col);
                     });

      // create string table view from str_column_vec:
      //
      auto str_table_ptr  = std::make_unique<cudf::table>(std::move(str_column_vec));
      auto str_table_view = str_table_ptr->view();

      // concatenate columns in each row into one big string column
      //(using null representation and delimiter):
      //
      std::string delimiter_str{options_.get_inter_column_delimiter()};
      auto str_concat_col =
        cudf::strings::concatenate(str_table_view, delimiter_str, options_.get_na_rep(), mr_);

      write_chunked(str_concat_col->view(), metadata, stream);
    }
  }

  // finalize (no-op, for now, but offers a hook for future extensions):
  //
  write_chunked_end(table, metadata, stream);
}

void writer::write(table_view const& table,
                   const table_metadata* metadata,
                   rmm::cuda_stream_view stream)
{
  _impl->write(table, metadata, stream);
}

}  // namespace csv
}  // namespace detail
}  // namespace io
}  // namespace cudf
