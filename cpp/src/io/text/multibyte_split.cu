#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2021-2022, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <io/utilities/output_builder.cuh>

#include <cudf/column/column.hpp>
#include <cudf/column/column_factories.hpp>
#include <cudf/detail/iterator.cuh>
#include <cudf/detail/nvtx/ranges.hpp>
#include <cudf/detail/utilities/integer_utils.hpp>
#include <cudf/io/text/byte_range_info.hpp>
#include <cudf/io/text/data_chunk_source.hpp>
#include <cudf/io/text/detail/multistate.hpp>
#include <cudf/io/text/detail/tile_state.hpp>
#include <cudf/io/text/multibyte_split.hpp>
#include <cudf/scalar/scalar.hpp>
#include <cudf/strings/detail/strings_column_factories.cuh>
#include <cudf/utilities/default_stream.hpp>
#include <cudf/utilities/span.hpp>

#include <rmm/cuda_stream_pool.hpp>
#include <rmm/cuda_stream_view.hpp>
#include <rmm/exec_policy.hpp>
#include <rmm/mr/device/device_memory_resource.hpp>
#include <rmm/mr/device/per_device_resource.hpp>

#include <thrust/copy.h>
#include <thrust/find.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/transform.h>

#include <cub/block/block_load.cuh>
#include <cub/block/block_scan.cuh>

#include <cstdint>
#include <limits>
#include <memory>
#include <numeric>
#include <optional>

namespace {

using cudf::io::text::detail::multistate;

int32_t constexpr ITEMS_PER_THREAD = 64;
int32_t constexpr THREADS_PER_TILE = 128;
int32_t constexpr ITEMS_PER_TILE   = ITEMS_PER_THREAD * THREADS_PER_TILE;
int32_t constexpr TILES_PER_CHUNK  = 4096;
int32_t constexpr ITEMS_PER_CHUNK  = ITEMS_PER_TILE * TILES_PER_CHUNK;

constexpr multistate transition_init(char c, cudf::device_span<char const> delim)
{
  auto result = multistate();

  result.enqueue(0, 0);

  for (std::size_t i = 0; i < delim.size(); i++) {
    if (delim[i] == c) { result.enqueue(i, i + 1); }
  }

  return result;
}

constexpr multistate transition(char c, multistate state, cudf::device_span<char const> delim)
{
  auto result = multistate();

  result.enqueue(0, 0);

  for (uint8_t i = 0; i < state.size(); i++) {
    auto const tail = state.get_tail(i);
    if (tail < delim.size() && delim[tail] == c) { result.enqueue(state.get_head(i), tail + 1); }
  }

  return result;
}

struct PatternScan {
  using BlockScan         = hipcub::BlockScan<multistate, THREADS_PER_TILE>;
  using BlockScanCallback = cudf::io::text::detail::scan_tile_state_callback<multistate>;

  struct _TempStorage {
    typename BlockScan::TempStorage scan;
  };

  _TempStorage& _temp_storage;

  using TempStorage = hipcub::Uninitialized<_TempStorage>;

  __device__ inline PatternScan(TempStorage& temp_storage) : _temp_storage(temp_storage.Alias()) {}

  __device__ inline void Scan(cudf::size_type tile_idx,
                              cudf::io::text::detail::scan_tile_state_view<multistate> tile_state,
                              cudf::device_span<char const> delim,
                              char (&thread_data)[ITEMS_PER_THREAD],
                              multistate& thread_multistate)
  {
    thread_multistate = transition_init(thread_data[0], delim);

    for (uint32_t i = 1; i < ITEMS_PER_THREAD; i++) {
      thread_multistate = transition(thread_data[i], thread_multistate, delim);
    }

    auto prefix_callback = BlockScanCallback(tile_state, tile_idx);

    BlockScan(_temp_storage.scan)
      .ExclusiveSum(thread_multistate, thread_multistate, prefix_callback);
  }
};

// type aliases to distinguish between row offsets and character offsets
using output_offset = int64_t;
using byte_offset   = int64_t;

// multibyte_split works by splitting up inputs in to 32 inputs (bytes) per thread, and transforming
// them in to data structures called "multistates". these multistates are created by searching a
// trie, but instead of a tradition trie where the search begins at a single node at the beginning,
// we allow our search to begin anywhere within the trie tree. The position within the trie tree is
// stored as a "partial match path", which indicates "we can get from here to there by a set of
// specific transitions". By scanning together multistates, we effectively know "we can get here
// from the beginning by following the inputs". By doing this, each thread knows exactly what state
// it begins in. From there, each thread can then take deterministic action. In this case, the
// deterministic action is counting and outputting delimiter offsets when a delimiter is found.

__global__ void multibyte_split_init_kernel(
  cudf::size_type base_tile_idx,
  cudf::size_type num_tiles,
  cudf::io::text::detail::scan_tile_state_view<multistate> tile_multistates,
  cudf::io::text::detail::scan_tile_state_view<output_offset> tile_output_offsets,
  cudf::io::text::detail::scan_tile_status status =
    cudf::io::text::detail::scan_tile_status::invalid)
{
  auto const thread_idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (thread_idx < num_tiles) {
    auto const tile_idx = base_tile_idx + thread_idx;
    tile_multistates.set_status(tile_idx, status);
    tile_output_offsets.set_status(tile_idx, status);
  }
}

__global__ void multibyte_split_seed_kernel(
  cudf::io::text::detail::scan_tile_state_view<multistate> tile_multistates,
  cudf::io::text::detail::scan_tile_state_view<output_offset> tile_output_offsets,
  multistate tile_multistate_seed,
  output_offset tile_output_offset)
{
  auto const thread_idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (thread_idx == 0) {
    tile_multistates.set_inclusive_prefix(-1, tile_multistate_seed);
    tile_output_offsets.set_inclusive_prefix(-1, tile_output_offset);
  }
}

__global__ __launch_bounds__(THREADS_PER_TILE) void multibyte_split_kernel(
  cudf::size_type base_tile_idx,
  byte_offset base_input_offset,
  output_offset base_output_offset,
  cudf::io::text::detail::scan_tile_state_view<multistate> tile_multistates,
  cudf::io::text::detail::scan_tile_state_view<output_offset> tile_output_offsets,
  cudf::device_span<char const> delim,
  cudf::device_span<char const> chunk_input_chars,
  cudf::split_device_span<byte_offset> row_offsets)
{
  using InputLoad =
    hipcub::BlockLoad<char, THREADS_PER_TILE, ITEMS_PER_THREAD, hipcub::BLOCK_LOAD_WARP_TRANSPOSE>;
  using OffsetScan         = hipcub::BlockScan<output_offset, THREADS_PER_TILE>;
  using OffsetScanCallback = cudf::io::text::detail::scan_tile_state_callback<output_offset>;

  __shared__ union {
    typename InputLoad::TempStorage input_load;
    typename PatternScan::TempStorage pattern_scan;
    typename OffsetScan::TempStorage offset_scan;
  } temp_storage;

  int32_t const tile_idx            = base_tile_idx + blockIdx.x;
  int32_t const tile_input_offset   = blockIdx.x * ITEMS_PER_TILE;
  int32_t const thread_input_offset = tile_input_offset + threadIdx.x * ITEMS_PER_THREAD;
  int32_t const thread_input_size   = chunk_input_chars.size() - thread_input_offset;

  // STEP 1: Load inputs

  char thread_chars[ITEMS_PER_THREAD];

  InputLoad(temp_storage.input_load)
    .Load(chunk_input_chars.data() + tile_input_offset,
          thread_chars,
          chunk_input_chars.size() - tile_input_offset);

  // STEP 2: Scan inputs to determine absolute thread states

  multistate thread_multistate;

  __syncthreads();  // required before temp_memory re-use
  PatternScan(temp_storage.pattern_scan)
    .Scan(tile_idx, tile_multistates, delim, thread_chars, thread_multistate);

  // STEP 3: Flag matches

  output_offset thread_offset{};
  uint32_t thread_match_mask[(ITEMS_PER_THREAD + 31) / 32]{};

  for (int32_t i = 0; i < ITEMS_PER_THREAD; i++) {
    thread_multistate       = transition(thread_chars[i], thread_multistate, delim);
    auto const thread_state = thread_multistate.max_tail();
    auto const is_match     = i < thread_input_size and thread_state == delim.size();
    thread_match_mask[i / 32] |= uint32_t{is_match} << (i % 32);
    thread_offset += output_offset{is_match};
  }

  // STEP 4: Scan flags to determine absolute thread output offset

  auto prefix_callback = OffsetScanCallback(tile_output_offsets, tile_idx);

  __syncthreads();  // required before temp_memory re-use
  OffsetScan(temp_storage.offset_scan).ExclusiveSum(thread_offset, thread_offset, prefix_callback);

  // Step 5: Assign outputs from each thread using match offsets.

  for (int32_t i = 0; i < ITEMS_PER_THREAD; i++) {
    auto const is_match = (thread_match_mask[i / 32] >> (i % 32)) & 1u;
    if (is_match) {
      auto const match_end = base_input_offset + thread_input_offset + i + 1;
      row_offsets[thread_offset - base_output_offset] = match_end;
      thread_offset++;
    }
  }
}

__global__ __launch_bounds__(THREADS_PER_TILE) void byte_split_kernel(
  cudf::size_type base_tile_idx,
  byte_offset base_input_offset,
  output_offset base_output_offset,
  cudf::io::text::detail::scan_tile_state_view<output_offset> tile_output_offsets,
  char delim,
  cudf::device_span<char const> chunk_input_chars,
  cudf::split_device_span<byte_offset> row_offsets)
{
  using InputLoad =
    hipcub::BlockLoad<char, THREADS_PER_TILE, ITEMS_PER_THREAD, hipcub::BLOCK_LOAD_WARP_TRANSPOSE>;
  using OffsetScan         = hipcub::BlockScan<output_offset, THREADS_PER_TILE>;
  using OffsetScanCallback = cudf::io::text::detail::scan_tile_state_callback<output_offset>;

  __shared__ union {
    typename InputLoad::TempStorage input_load;
    typename OffsetScan::TempStorage offset_scan;
  } temp_storage;

  int32_t const tile_idx            = base_tile_idx + blockIdx.x;
  int32_t const tile_input_offset   = blockIdx.x * ITEMS_PER_TILE;
  int32_t const thread_input_offset = tile_input_offset + threadIdx.x * ITEMS_PER_THREAD;
  int32_t const thread_input_size   = chunk_input_chars.size() - thread_input_offset;

  // STEP 1: Load inputs

  char thread_chars[ITEMS_PER_THREAD];

  InputLoad(temp_storage.input_load)
    .Load(chunk_input_chars.data() + tile_input_offset,
          thread_chars,
          chunk_input_chars.size() - tile_input_offset);

  // STEP 2: Flag matches

  output_offset thread_offset{};
  uint32_t thread_match_mask[(ITEMS_PER_THREAD + 31) / 32]{};

  for (int32_t i = 0; i < ITEMS_PER_THREAD; i++) {
    auto const is_match = i < thread_input_size and thread_chars[i] == delim;
    thread_match_mask[i / 32] |= uint32_t{is_match} << (i % 32);
    thread_offset += output_offset{is_match};
  }

  // STEP 3: Scan flags to determine absolute thread output offset

  auto prefix_callback = OffsetScanCallback(tile_output_offsets, tile_idx);

  __syncthreads();  // required before temp_memory re-use
  OffsetScan(temp_storage.offset_scan).ExclusiveSum(thread_offset, thread_offset, prefix_callback);

  // Step 4: Assign outputs from each thread using match offsets.

  for (int32_t i = 0; i < ITEMS_PER_THREAD; i++) {
    auto const is_match = (thread_match_mask[i / 32] >> (i % 32)) & 1u;
    if (is_match) {
      auto const match_end = base_input_offset + thread_input_offset + i + 1;
      row_offsets[thread_offset - base_output_offset] = match_end;
      thread_offset++;
    }
  }
}

}  // namespace

namespace cudf {
namespace io {
namespace text {
namespace detail {

void fork_stream(std::vector<rmm::cuda_stream_view> streams, rmm::cuda_stream_view stream)
{
  hipEvent_t event;
  hipEventCreate(&event);
  hipEventRecord(event, stream);
  for (uint32_t i = 0; i < streams.size(); i++) {
    hipStreamWaitEvent(streams[i], event, 0);
  }
  hipEventDestroy(event);
}

void join_stream(std::vector<rmm::cuda_stream_view> streams, rmm::cuda_stream_view stream)
{
  hipEvent_t event;
  hipEventCreate(&event);
  for (uint32_t i = 0; i < streams.size(); i++) {
    hipEventRecord(event, streams[i]);
    hipStreamWaitEvent(stream, event, 0);
  }
  hipEventDestroy(event);
}

std::vector<rmm::cuda_stream_view> get_streams(int32_t count, rmm::cuda_stream_pool& stream_pool)
{
  auto streams = std::vector<rmm::cuda_stream_view>();
  for (int32_t i = 0; i < count; i++) {
    streams.emplace_back(stream_pool.get_stream());
  }
  return streams;
}

std::unique_ptr<cudf::column> multibyte_split(cudf::io::text::data_chunk_source const& source,
                                              std::string const& delimiter,
                                              byte_range_info byte_range,
                                              bool strip_delimiters,
                                              rmm::cuda_stream_view stream,
                                              rmm::mr::device_memory_resource* mr,
                                              rmm::cuda_stream_pool& stream_pool)
{
  CUDF_FUNC_RANGE();

  if (byte_range.empty()) { return make_empty_column(type_id::STRING); }

  auto device_delim = cudf::string_scalar(delimiter, true, stream, mr);

  auto sorted_delim = delimiter;
  std::sort(sorted_delim.begin(), sorted_delim.end());
  auto [_last_char, _last_char_count, max_duplicate_tokens] = std::accumulate(
    sorted_delim.begin(), sorted_delim.end(), std::make_tuple('\0', 0, 0), [](auto acc, char c) {
      if (std::get<0>(acc) != c) {
        std::get<0>(acc) = c;
        std::get<1>(acc) = 0;
      }
      std::get<1>(acc)++;
      std::get<2>(acc) = std::max(std::get<1>(acc), std::get<2>(acc));
      return acc;
    });

  CUDF_EXPECTS(max_duplicate_tokens < multistate::max_segment_count,
               "delimiter contains too many duplicate tokens to produce a deterministic result.");

  CUDF_EXPECTS(delimiter.size() < multistate::max_segment_value,
               "delimiter contains too many total tokens to produce a deterministic result.");

  auto concurrency = 2;
  auto streams     = get_streams(concurrency, stream_pool);

  // must be at least 32 when using warp-reduce on partials
  // must be at least 1 more than max possible concurrent tiles
  // best when at least 32 more than max possible concurrent tiles, due to rolling `invalid`s
  auto num_tile_states  = std::max(32, TILES_PER_CHUNK * concurrency + 32);
  auto tile_multistates = scan_tile_state<multistate>(num_tile_states, stream);
  auto tile_offsets     = scan_tile_state<output_offset>(num_tile_states, stream);

  multibyte_split_init_kernel<<<TILES_PER_CHUNK,
                                THREADS_PER_TILE,
                                0,
                                stream.value()>>>(  //
    -TILES_PER_CHUNK,
    TILES_PER_CHUNK,
    tile_multistates,
    tile_offsets,
    cudf::io::text::detail::scan_tile_status::oob);

  auto multistate_seed = multistate();
  multistate_seed.enqueue(0, 0);  // this represents the first state in the pattern.

  // Seeding the tile state with an identity value allows the 0th tile to follow the same logic as
  // the Nth tile, assuming it can look up an inclusive prefix. Without this seed, the 0th block
  // would have to follow separate logic.
  multibyte_split_seed_kernel<<<1, 1, 0, stream.value()>>>(  //
    tile_multistates,
    tile_offsets,
    multistate_seed,
    0);

  auto reader               = source.create_reader();
  auto chunk_offset         = std::max<byte_offset>(0, byte_range.offset() - delimiter.size());
  auto const byte_range_end = byte_range.offset() + byte_range.size();
  reader->skip_bytes(chunk_offset);
  // amortize output chunk allocations over 8 worst-case outputs. This limits the overallocation
  constexpr auto max_growth = 8;
  output_builder<byte_offset> row_offset_storage(ITEMS_PER_CHUNK, max_growth, stream);
  output_builder<char> char_storage(ITEMS_PER_CHUNK, max_growth, stream);

  fork_stream(streams, stream);

  hipEvent_t last_launch_event;
  hipEventCreate(&last_launch_event);

  auto& read_stream     = streams[0];
  auto& scan_stream     = streams[1];
  auto chunk            = reader->get_next_chunk(ITEMS_PER_CHUNK, read_stream);
  int64_t base_tile_idx = 0;
  std::optional<byte_offset> first_row_offset;
  std::optional<byte_offset> last_row_offset;
  bool found_last_offset = false;
  if (byte_range.offset() == 0) { first_row_offset = 0; }
  std::swap(read_stream, scan_stream);

  while (chunk->size() > 0) {
    // if we found the last delimiter, or didn't find delimiters inside the byte range at all: abort
    if (last_row_offset.has_value() or
        (not first_row_offset.has_value() and chunk_offset >= byte_range_end)) {
      break;
    }

    auto tiles_in_launch =
      cudf::util::div_rounding_up_safe(chunk->size(), static_cast<std::size_t>(ITEMS_PER_TILE));

    auto row_offsets = row_offset_storage.next_output(scan_stream);

    // reset the next chunk of tile state
    multibyte_split_init_kernel<<<tiles_in_launch,
                                  THREADS_PER_TILE,
                                  0,
                                  scan_stream.value()>>>(  //
      base_tile_idx,
      tiles_in_launch,
      tile_multistates,
      tile_offsets);

    hipStreamWaitEvent(scan_stream.value(), last_launch_event);

    if (delimiter.size() == 1) {
      // the single-byte case allows for a much more efficient kernel, so we special-case it
      byte_split_kernel<<<tiles_in_launch,
                          THREADS_PER_TILE,
                          0,
                          scan_stream.value()>>>(  //
        base_tile_idx,
        chunk_offset,
        row_offset_storage.size(),
        tile_offsets,
        delimiter[0],
        *chunk,
        row_offsets);
    } else {
      multibyte_split_kernel<<<tiles_in_launch,
                               THREADS_PER_TILE,
                               0,
                               scan_stream.value()>>>(  //
        base_tile_idx,
        chunk_offset,
        row_offset_storage.size(),
        tile_multistates,
        tile_offsets,
        {device_delim.data(), static_cast<std::size_t>(device_delim.size())},
        *chunk,
        row_offsets);
    }

    // load the next chunk
    auto next_chunk = reader->get_next_chunk(ITEMS_PER_CHUNK, read_stream);
    // while that is running, determine how many offsets we output (synchronizes)
    auto const new_offsets = [&] {
      auto const new_offsets_unclamped =
        tile_offsets.get_inclusive_prefix(base_tile_idx + tiles_in_launch - 1, scan_stream) -
        static_cast<output_offset>(row_offset_storage.size());
      // if we are not in the last chunk, we can use all offsets
      if (chunk_offset + static_cast<output_offset>(chunk->size()) < byte_range_end) {
        return new_offsets_unclamped;
      }
      // if we are in the last chunk, we need to find the first out-of-bounds offset
      auto const it = thrust::make_counting_iterator(output_offset{});
      auto const end_loc =
        *thrust::find_if(rmm::exec_policy_nosync(scan_stream),
                         it,
                         it + new_offsets_unclamped,
                         [row_offsets, byte_range_end] __device__(output_offset i) {
                           return row_offsets[i] >= byte_range_end;
                         });
      // if we had no out-of-bounds offset, we copy all offsets
      if (end_loc == new_offsets_unclamped) { return end_loc; }
      // otherwise we copy only up to (including) the first out-of-bounds delimiter
      found_last_offset = true;
      return end_loc + 1;
    }();
    row_offset_storage.advance_output(new_offsets, scan_stream);
    // determine if we found the first or last field offset for the byte range
    if (new_offsets > 0 and not first_row_offset) {
      first_row_offset = row_offset_storage.front_element(scan_stream);
    }
    if (found_last_offset) { last_row_offset = row_offset_storage.back_element(scan_stream); }
    // copy over the characters we need, if we already encountered the first field delimiter
    if (first_row_offset.has_value()) {
      auto const begin = chunk->data() + std::max<byte_offset>(0, *first_row_offset - chunk_offset);
      auto const sentinel = last_row_offset.value_or(std::numeric_limits<byte_offset>::max());
      auto const end =
        chunk->data() + std::min<byte_offset>(sentinel - chunk_offset, chunk->size());
      auto const output_size = end - begin;
      auto char_output       = char_storage.next_output(scan_stream);
      thrust::copy(rmm::exec_policy_nosync(scan_stream), begin, end, char_output.begin());
      char_storage.advance_output(output_size, scan_stream);
    }

    hipEventRecord(last_launch_event, scan_stream.value());

    std::swap(read_stream, scan_stream);
    base_tile_idx += tiles_in_launch;
    chunk_offset += chunk->size();
    chunk = std::move(next_chunk);
  }

  hipEventDestroy(last_launch_event);

  join_stream(streams, stream);

  // if the input was empty, we didn't find a delimiter at all,
  // or the first delimiter was also the last: empty output
  if (chunk_offset == 0 or not first_row_offset.has_value() or
      first_row_offset == last_row_offset) {
    return make_empty_column(type_id::STRING);
  }

  auto chars          = char_storage.gather(stream, mr);
  auto global_offsets = row_offset_storage.gather(stream, mr);

  // insert an offset at the beginning if we started at the beginning of the input
  bool const insert_begin = first_row_offset.value_or(0) == 0;
  // insert an offset at the end if we have not terminated the last row
  bool const insert_end =
    not(last_row_offset.has_value() or
        (global_offsets.size() > 0 and global_offsets.back_element(stream) == chunk_offset));
  rmm::device_uvector<int32_t> offsets{
    global_offsets.size() + insert_begin + insert_end, stream, mr};
  if (insert_begin) { offsets.set_element_to_zero_async(0, stream); }
  if (insert_end) {
    offsets.set_element(offsets.size() - 1, chunk_offset - *first_row_offset, stream);
  }
  thrust::transform(rmm::exec_policy(stream),
                    global_offsets.begin(),
                    global_offsets.end(),
                    offsets.begin() + insert_begin,
                    [baseline = *first_row_offset] __device__(byte_offset global_offset) {
                      return static_cast<int32_t>(global_offset - baseline);
                    });
  auto string_count = offsets.size() - 1;
  if (strip_delimiters) {
    auto it = cudf::detail::make_counting_transform_iterator(
      0,
      [ofs        = offsets.data(),
       chars      = chars.data(),
       delim_size = static_cast<size_type>(delimiter.size()),
       last_row   = static_cast<size_type>(string_count) - 1,
       insert_end] __device__(size_type row) {
        auto const begin = ofs[row];
        auto const len   = ofs[row + 1] - begin;
        if (row == last_row && insert_end) {
          return thrust::make_pair(chars + begin, len);
        } else {
          return thrust::make_pair(chars + begin, std::max<size_type>(0, len - delim_size));
        };
      });
    return cudf::strings::detail::make_strings_column(it, it + string_count, stream, mr);
  } else {
    return cudf::make_strings_column(string_count, std::move(offsets), std::move(chars));
  }
}

}  // namespace detail

std::unique_ptr<cudf::column> multibyte_split(cudf::io::text::data_chunk_source const& source,
                                              std::string const& delimiter,
                                              std::optional<byte_range_info> byte_range,
                                              rmm::mr::device_memory_resource* mr)
{
  return multibyte_split(
    source, delimiter, parse_options{byte_range.value_or(create_byte_range_info_max())}, mr);
}

std::unique_ptr<cudf::column> multibyte_split(cudf::io::text::data_chunk_source const& source,
                                              std::string const& delimiter,
                                              parse_options options,
                                              rmm::mr::device_memory_resource* mr)
{
  auto stream      = cudf::get_default_stream();
  auto stream_pool = rmm::cuda_stream_pool(2);

  auto result = detail::multibyte_split(
    source, delimiter, options.byte_range, options.strip_delimiters, stream, mr, stream_pool);

  return result;
}

std::unique_ptr<cudf::column> multibyte_split(cudf::io::text::data_chunk_source const& source,
                                              std::string const& delimiter,
                                              rmm::mr::device_memory_resource* mr)
{
  return multibyte_split(source, delimiter, parse_options{}, mr);
}

}  // namespace text
}  // namespace io
}  // namespace cudf
