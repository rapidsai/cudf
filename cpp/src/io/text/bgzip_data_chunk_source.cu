#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2022, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "io/comp/nvcomp_adapter.hpp"

#include <cudf/detail/nvtx/ranges.hpp>
#include <cudf/io/text/data_chunk_source_factories.hpp>
#include <cudf/utilities/default_stream.hpp>
#include <cudf/utilities/error.hpp>

#include <rmm/cuda_stream_view.hpp>
#include <rmm/device_buffer.hpp>
#include <rmm/exec_policy.hpp>

#include <thrust/host_vector.h>
#include <thrust/iterator/zip_iterator.h>
#include <thrust/system/cuda/experimental/pinned_allocator.h>
#include <thrust/transform.h>

#include <fstream>
#include <limits>

namespace cudf::io::text {

namespace {

class device_uvector_data_chunk : public device_data_chunk {
 public:
  device_uvector_data_chunk(rmm::device_uvector<char>&& data) : _data(std::move(data)) {}

  [[nodiscard]] char const* data() const override { return _data.data(); }
  [[nodiscard]] std::size_t size() const override { return _data.size(); }
  operator device_span<char const>() const override { return _data; }

 private:
  rmm::device_uvector<char> _data;
};

struct bgzip_nvcomp_transform_functor {
  uint8_t const* compressed_ptr;
  uint8_t* decompressed_ptr;

  __device__ thrust::tuple<device_span<const uint8_t>, device_span<uint8_t>> operator()(
    thrust::tuple<std::size_t, std::size_t, std::size_t, std::size_t> t)
  {
    auto const compressed_begin   = thrust::get<0>(t);
    auto const compressed_end     = thrust::get<1>(t);
    auto const decompressed_begin = thrust::get<2>(t);
    auto const decompressed_end   = thrust::get<3>(t);
    return thrust::make_tuple(device_span<const uint8_t>{compressed_ptr + compressed_begin,
                                                         compressed_end - compressed_begin},
                              device_span<uint8_t>{decompressed_ptr + decompressed_begin,
                                                   decompressed_end - decompressed_begin});
  }
};

class bgzip_data_chunk_reader : public data_chunk_reader {
 private:
  /*


     Parsing code


  */

  template <typename IntType>
  static bool is_little_endian()
  {
    IntType i = 0x0100;
    std::array<char, sizeof(i)> bytes;
    std::memcpy(&bytes[0], &i, sizeof(i));
    return bytes[0] == 0;
  }

  template <typename IntType>
  static IntType read_int(char* data)
  {
    IntType result{};
    if (not is_little_endian<IntType>()) { std::reverse(data, data + sizeof(result)); }
    std::memcpy(&result, &data[0], sizeof(result));
    return result;
  }

  struct bgzip_header {
    int block_size;
    int extra_length;
    [[nodiscard]] int data_size() const { return block_size - extra_length - 20; }
  };

  bgzip_header read_header()
  {
    std::array<char, 12> buffer{};
    _stream->read(buffer.data(), sizeof(buffer));
    uint8_t constexpr magic_expected_2 = 139;
    std::array<char, 4> expected_header{{31, 0, 8, 4}};
    std::memcpy(&expected_header[1], &magic_expected_2, sizeof(char));
    CUDF_EXPECTS(std::equal(expected_header.begin(), expected_header.end(), buffer.begin()),
                 "malformed BGZIP header");
    auto extra_length = read_int<uint16_t>(&buffer[10]);
    uint16_t extra_offset{};
    // read all the extra subfields
    while (extra_offset < extra_length) {
      auto const remaining_size = extra_length - extra_offset;
      CUDF_EXPECTS(remaining_size >= 4, "invalid extra field length");
      _stream->read(buffer.data(), 4);
      extra_offset += 4;
      auto subfield_size = read_int<uint16_t>(&buffer[2]);
      if (buffer[0] == 66 && buffer[1] == 67) {
        CUDF_EXPECTS(subfield_size == sizeof(uint16_t), "malformed BGZIP extra subfield");
        _stream->read(buffer.data(), sizeof(uint16_t));
        _stream->seekg(remaining_size - 6, std::ios_base::cur);
        auto block_size = read_int<uint16_t>(&buffer[0]);
        return {block_size + 1, extra_length};
      } else {
        _stream->seekg(subfield_size, std::ios_base::cur);
        extra_offset += subfield_size;
      }
    }
    CUDF_FAIL("missing BGZIP size extra subfield");
  }

  struct bgzip_footer {
    uint32_t decompressed_size;
  };

  bgzip_footer read_footer()
  {
    std::array<char, 8> buffer{};
    _stream->read(buffer.data(), sizeof(buffer));
    return {read_int<uint32_t>(&buffer[4])};
  }

  /*


    High-level control flow


  */

  template <typename T>
  using pinned_host_vector =
    thrust::host_vector<T, thrust::system::cuda::experimental::pinned_allocator<T>>;

  template <typename T>
  static void copy_to_device(const pinned_host_vector<T>& host,
                             rmm::device_uvector<T>& device,
                             rmm::cuda_stream_view stream)
  {
    device.resize(host.size(), stream);
    CUDF_CUDA_TRY(hipMemcpyAsync(
      device.data(), host.data(), host.size() * sizeof(T), hipMemcpyHostToDevice, stream.value()));
  }

  struct compressed_blocks {
    static constexpr std::size_t default_buffer_alloc =
      1 << 24;  // 16MB buffer allocation, resized on demand
    static constexpr std::size_t default_offset_alloc =
      1 << 16;  // 64k offset allocation, resized on demand

    hipEvent_t event;
    pinned_host_vector<char> h_compressed_blocks;
    pinned_host_vector<std::size_t> h_compressed_offsets;
    pinned_host_vector<std::size_t> h_decompressed_offsets;
    rmm::device_uvector<char> d_compressed_blocks;
    rmm::device_uvector<char> d_decompressed_blocks;
    rmm::device_uvector<std::size_t> d_compressed_offsets;
    rmm::device_uvector<std::size_t> d_decompressed_offsets;
    rmm::device_uvector<device_span<const uint8_t>> d_compressed_spans;
    rmm::device_uvector<device_span<uint8_t>> d_decompressed_spans;
    rmm::device_uvector<compression_result> d_decompression_results;
    std::size_t compressed_size_with_headers{};
    std::size_t max_decompressed_size{};
    // this is usually equal to decompressed_size()
    // unless we are in the last chunk, where it's limited by _local_end
    std::size_t available_decompressed_size{};
    std::size_t read_pos{};
    bool decompressed{};

    compressed_blocks()
      : d_compressed_blocks(0, cudf::default_stream_value),
        d_decompressed_blocks(0, cudf::default_stream_value),
        d_compressed_offsets(0, cudf::default_stream_value),
        d_decompressed_offsets(0, cudf::default_stream_value),
        d_compressed_spans(0, cudf::default_stream_value),
        d_decompressed_spans(0, cudf::default_stream_value),
        d_decompression_results(0, cudf::default_stream_value)
    {
      CUDF_CUDA_TRY(hipEventCreate(&event));
      h_compressed_blocks.reserve(default_buffer_alloc);
      h_compressed_offsets.reserve(default_offset_alloc);
      h_compressed_offsets.push_back(0);
      h_decompressed_offsets.reserve(default_offset_alloc);
      h_decompressed_offsets.push_back(0);
    }

    void decompress(rmm::cuda_stream_view stream)
    {
      if (decompressed) { return; }
      copy_to_device(h_compressed_blocks, d_compressed_blocks, stream);
      copy_to_device(h_compressed_offsets, d_compressed_offsets, stream);
      copy_to_device(h_decompressed_offsets, d_decompressed_offsets, stream);
      d_decompressed_blocks.resize(decompressed_size(), stream);
      d_compressed_spans.resize(num_blocks(), stream);
      d_decompressed_spans.resize(num_blocks(), stream);
      d_decompression_results.resize(num_blocks(), stream);

      auto offset_it = thrust::make_zip_iterator(d_compressed_offsets.begin(),
                                                 d_compressed_offsets.begin() + 1,
                                                 d_decompressed_offsets.begin(),
                                                 d_decompressed_offsets.begin() + 1);
      auto span_it =
        thrust::make_zip_iterator(d_compressed_spans.begin(), d_decompressed_spans.begin());
      thrust::transform(
        rmm::exec_policy_nosync(stream),
        offset_it,
        offset_it + num_blocks(),
        span_it,
        bgzip_nvcomp_transform_functor{reinterpret_cast<uint8_t*>(d_compressed_blocks.data()),
                                       reinterpret_cast<uint8_t*>(d_decompressed_blocks.begin())});
      if (decompressed_size() > 0) {
        cudf::io::nvcomp::batched_decompress(cudf::io::nvcomp::compression_type::DEFLATE,
                                             d_compressed_spans,
                                             d_decompressed_spans,
                                             d_decompression_results,
                                             max_decompressed_size,
                                             decompressed_size(),
                                             stream);
      }
      decompressed = true;
    }

    void reset()
    {
      h_compressed_blocks.resize(0);
      h_compressed_offsets.resize(1);
      h_decompressed_offsets.resize(1);
      // shrinking doesn't allocate/free, so we don't need to worry about streams
      auto stream = cudf::default_stream_value;
      d_compressed_blocks.resize(0, stream);
      d_decompressed_blocks.resize(0, stream);
      d_compressed_offsets.resize(0, stream);
      d_decompressed_offsets.resize(0, stream);
      d_compressed_spans.resize(0, stream);
      d_decompressed_spans.resize(0, stream);
      d_decompression_results.resize(0, stream);
      compressed_size_with_headers = 0;
      max_decompressed_size        = 0;
      available_decompressed_size  = 0;
      read_pos                     = 0;
      decompressed                 = false;
    }

    [[nodiscard]] std::size_t num_blocks() const { return h_compressed_offsets.size() - 1; }

    [[nodiscard]] std::size_t compressed_size() const { return h_compressed_offsets.back(); }

    [[nodiscard]] std::size_t decompressed_size() const { return h_decompressed_offsets.back(); }

    [[nodiscard]] std::size_t remaining_size() const
    {
      return available_decompressed_size - read_pos;
    }

    void read_block(bgzip_header header, std::istream& stream)
    {
      h_compressed_blocks.resize(h_compressed_blocks.size() + header.data_size());
      stream.read(h_compressed_blocks.data() + compressed_size(), header.data_size());
    }

    void add_block_offsets(bgzip_header header, bgzip_footer footer)
    {
      max_decompressed_size =
        std::max<std::size_t>(footer.decompressed_size, max_decompressed_size);
      h_compressed_offsets.push_back(compressed_size() + header.data_size());
      h_decompressed_offsets.push_back(decompressed_size() + footer.decompressed_size);
    }

    void consume_bytes(std::size_t size)
    {
      CUDF_EXPECTS(size <= remaining_size(), "out of bounds");
      read_pos += size;
    }
  };

  void read_next_compressed_chunk(std::size_t requested_size)
  {
    std::swap(_curr_block, _prev_block);
    if (_curr_block.decompressed) {
      // synchronize on the last decompression + copy, so we don't clobber any buffers
      CUDF_CUDA_TRY(hipEventSynchronize(_curr_block.event));
    }
    _curr_block.reset();
    // read chunks until we have enough decompressed data
    while (_curr_block.decompressed_size() < requested_size) {
      _stream->peek();
      if (_stream->eof() || _compressed_pos > _compressed_end) { break; }
      auto header = read_header();
      _curr_block.read_block(header, *_stream);
      auto footer = read_footer();
      _curr_block.add_block_offsets(header, footer);
      // for the last GZIP block, we restrict ourselves to the bytes up to _local_end
      // but only for the reader, not for decompression!
      if (_compressed_pos == _compressed_end) {
        _curr_block.available_decompressed_size += _local_end;
        _compressed_pos += header.block_size;
        break;
      } else {
        _curr_block.available_decompressed_size += footer.decompressed_size;
        _compressed_pos += header.block_size;
      }
    }
  }

  constexpr static std::size_t chunk_load_size = 1 << 24;  // load 16 MB of data by default

 public:
  bgzip_data_chunk_reader(std::unique_ptr<std::istream> input_stream,
                          uint64_t virtual_begin,
                          uint64_t virtual_end)
    : _stream(std::move(input_stream)),
      _local_end{virtual_end & 0xFFFFu},
      _compressed_pos{virtual_begin >> 16},
      _compressed_end{virtual_end >> 16}
  {
    // set failbit to throw on IO failures
    input_stream->exceptions(input_stream->exceptions() | std::istream::failbit);
    // seek to the beginning of the provided compressed offset
    _stream->seekg(_compressed_pos, std::ios_base::cur);
    // read the first blocks
    read_next_compressed_chunk(chunk_load_size);
    // seek to the beginning of the provided local offset
    auto const local_pos = virtual_begin & 0xFFFFu;
    if (local_pos > 0) {
      CUDF_EXPECTS(_curr_block.h_compressed_offsets.size() > 1 &&
                     local_pos < _curr_block.h_compressed_offsets[1],
                   "local part of virtual offset is out of bounds");
      _curr_block.consume_bytes(local_pos);
    }
  }

  void skip_bytes(std::size_t read_size) override
  {
    while (read_size > _curr_block.remaining_size()) {
      read_size -= _curr_block.remaining_size();
      _curr_block.consume_bytes(_curr_block.remaining_size());
      read_next_compressed_chunk(chunk_load_size);
      _stream->peek();
      if (_stream->eof() || _compressed_pos > _compressed_end) { break; }
    }
    read_size = std::min(read_size, _curr_block.remaining_size());
    _curr_block.consume_bytes(read_size);
  }

  std::unique_ptr<device_data_chunk> get_next_chunk(std::size_t read_size,
                                                    rmm::cuda_stream_view hip_stream) override
  {
    CUDF_FUNC_RANGE();
    if (read_size <= _curr_block.remaining_size()) {
      _curr_block.decompress(hip_stream);
      rmm::device_uvector<char> data(read_size, hip_stream);
      CUDF_CUDA_TRY(hipMemcpyAsync(data.data(),
                                    _curr_block.d_decompressed_blocks.data() + _curr_block.read_pos,
                                    read_size,
                                    hipMemcpyDeviceToDevice,
                                    hip_stream.value()));
      // record the host-to-device copy, decompression and device copy
      CUDF_CUDA_TRY(hipEventRecord(_curr_block.event, hip_stream.value()));
      _curr_block.consume_bytes(read_size);
      return std::make_unique<device_uvector_data_chunk>(std::move(data));
    }
    read_next_compressed_chunk(read_size /* - _curr_block.remaining_size()*/);
    _prev_block.decompress(hip_stream);
    _curr_block.decompress(hip_stream);
    read_size = std::min(read_size, _prev_block.remaining_size() + _curr_block.remaining_size());
    rmm::device_uvector<char> data(read_size, hip_stream);
    CUDF_CUDA_TRY(hipMemcpyAsync(data.data(),
                                  _prev_block.d_decompressed_blocks.data() + _prev_block.read_pos,
                                  _prev_block.remaining_size(),
                                  hipMemcpyDeviceToDevice,
                                  hip_stream.value()));
    CUDF_CUDA_TRY(hipMemcpyAsync(data.data() + _prev_block.remaining_size(),
                                  _curr_block.d_decompressed_blocks.data() + _curr_block.read_pos,
                                  read_size - _prev_block.remaining_size(),
                                  hipMemcpyDeviceToDevice,
                                  hip_stream.value()));
    // record the host-to-device copy, decompression and device copy
    CUDF_CUDA_TRY(hipEventRecord(_curr_block.event, hip_stream.value()));
    CUDF_CUDA_TRY(hipEventRecord(_prev_block.event, hip_stream.value()));
    read_size -= _prev_block.remaining_size();
    _prev_block.consume_bytes(_prev_block.remaining_size());
    _curr_block.consume_bytes(read_size);
    return std::make_unique<device_uvector_data_chunk>(std::move(data));
  }

 private:
  std::unique_ptr<std::istream> _stream;
  compressed_blocks _prev_block;
  compressed_blocks _curr_block;
  std::size_t _local_end;
  std::size_t _compressed_pos;
  std::size_t _compressed_end;
};

class bgzip_data_chunk_source : public data_chunk_source {
 public:
  bgzip_data_chunk_source(std::string filename, uint64_t virtual_begin, uint64_t virtual_end)
    : _filename{std::move(filename)}, _virtual_begin{virtual_begin}, _virtual_end{virtual_end}
  {
  }
  [[nodiscard]] std::unique_ptr<data_chunk_reader> create_reader() const override
  {
    return std::make_unique<bgzip_data_chunk_reader>(
      std::make_unique<std::ifstream>(_filename, std::ifstream::in), _virtual_begin, _virtual_end);
  }

 private:
  std::string _filename;
  uint64_t _virtual_begin;
  uint64_t _virtual_end;
};

}  // namespace

std::unique_ptr<data_chunk_source> make_source_from_bgzip_file(std::string const& filename,
                                                               uint64_t virtual_begin,
                                                               uint64_t virtual_end)
{
  return std::make_unique<bgzip_data_chunk_source>(filename, virtual_begin, virtual_end);
}

std::unique_ptr<data_chunk_source> make_source_from_bgzip_file(std::string const& filename)
{
  return std::make_unique<bgzip_data_chunk_source>(
    filename, 0, std::numeric_limits<uint64_t>::max());
}

}  // namespace cudf::io::text
