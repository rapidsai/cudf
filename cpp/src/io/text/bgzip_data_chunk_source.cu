#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2022, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "io/comp/nvcomp_adapter.hpp"
#include "io/text/device_data_chunks.hpp"
#include "io/utilities/config_utils.hpp"

#include <cudf/detail/nvtx/ranges.hpp>
#include <cudf/detail/utilities/pinned_allocator.hpp>
#include <cudf/io/text/data_chunk_source_factories.hpp>
#include <cudf/io/text/detail/bgzip_utils.hpp>
#include <cudf/utilities/default_stream.hpp>
#include <cudf/utilities/error.hpp>

#include <rmm/cuda_stream_view.hpp>
#include <rmm/device_buffer.hpp>
#include <rmm/exec_policy.hpp>

#include <thrust/host_vector.h>
#include <thrust/iterator/zip_iterator.h>
#include <thrust/transform.h>

#include <fstream>
#include <limits>

namespace cudf::io::text {
namespace {

/**
 * @brief Transforms offset tuples of the form [compressed_begin, compressed_end,
 * decompressed_begin, decompressed_end] into span tuples of the form [compressed_device_span,
 * decompressed_device_span] based on the provided pointers.
 */
struct bgzip_nvcomp_transform_functor {
  uint8_t const* compressed_ptr;
  uint8_t* decompressed_ptr;

  __device__ thrust::tuple<device_span<const uint8_t>, device_span<uint8_t>> operator()(
    thrust::tuple<std::size_t, std::size_t, std::size_t, std::size_t> t)
  {
    auto const compressed_begin   = thrust::get<0>(t);
    auto const compressed_end     = thrust::get<1>(t);
    auto const decompressed_begin = thrust::get<2>(t);
    auto const decompressed_end   = thrust::get<3>(t);
    return thrust::make_tuple(device_span<const uint8_t>{compressed_ptr + compressed_begin,
                                                         compressed_end - compressed_begin},
                              device_span<uint8_t>{decompressed_ptr + decompressed_begin,
                                                   decompressed_end - decompressed_begin});
  }
};

class bgzip_data_chunk_reader : public data_chunk_reader {
 private:
  template <typename T>
  using pinned_host_vector = thrust::host_vector<T, cudf::detail::pinned_allocator<T>>;

  template <typename T>
  static void copy_to_device(const pinned_host_vector<T>& host,
                             rmm::device_uvector<T>& device,
                             rmm::cuda_stream_view stream)
  {
    device.resize(host.size(), stream);
    CUDF_CUDA_TRY(hipMemcpyAsync(
      device.data(), host.data(), host.size() * sizeof(T), hipMemcpyHostToDevice, stream.value()));
  }

  struct decompression_blocks {
    static constexpr std::size_t default_buffer_alloc =
      1 << 24;  // 16MB buffer allocation, resized on demand
    static constexpr std::size_t default_offset_alloc =
      1 << 16;  // 64k offset allocation, resized on demand

    hipEvent_t event;
    pinned_host_vector<char> h_compressed_blocks;
    pinned_host_vector<std::size_t> h_compressed_offsets;
    pinned_host_vector<std::size_t> h_decompressed_offsets;
    rmm::device_uvector<char> d_compressed_blocks;
    rmm::device_uvector<char> d_decompressed_blocks;
    rmm::device_uvector<std::size_t> d_compressed_offsets;
    rmm::device_uvector<std::size_t> d_decompressed_offsets;
    rmm::device_uvector<device_span<const uint8_t>> d_compressed_spans;
    rmm::device_uvector<device_span<uint8_t>> d_decompressed_spans;
    rmm::device_uvector<compression_result> d_decompression_results;
    std::size_t compressed_size_with_headers{};
    std::size_t max_decompressed_size{};
    // this is usually equal to decompressed_size()
    // unless we are in the last chunk, where it's limited by _local_end
    std::size_t available_decompressed_size{};
    std::size_t read_pos{};
    bool is_decompressed{};

    decompression_blocks(rmm::cuda_stream_view init_stream)
      : d_compressed_blocks(0, init_stream),
        d_decompressed_blocks(0, init_stream),
        d_compressed_offsets(0, init_stream),
        d_decompressed_offsets(0, init_stream),
        d_compressed_spans(0, init_stream),
        d_decompressed_spans(0, init_stream),
        d_decompression_results(0, init_stream)
    {
      CUDF_CUDA_TRY(hipEventCreate(&event));
      h_compressed_blocks.reserve(default_buffer_alloc);
      h_compressed_offsets.reserve(default_offset_alloc);
      h_compressed_offsets.push_back(0);
      h_decompressed_offsets.reserve(default_offset_alloc);
      h_decompressed_offsets.push_back(0);
    }

    void decompress(rmm::cuda_stream_view stream)
    {
      if (is_decompressed) { return; }
      copy_to_device(h_compressed_blocks, d_compressed_blocks, stream);
      copy_to_device(h_compressed_offsets, d_compressed_offsets, stream);
      copy_to_device(h_decompressed_offsets, d_decompressed_offsets, stream);
      d_decompressed_blocks.resize(decompressed_size(), stream);
      d_compressed_spans.resize(num_blocks(), stream);
      d_decompressed_spans.resize(num_blocks(), stream);
      d_decompression_results.resize(num_blocks(), stream);

      auto offset_it = thrust::make_zip_iterator(d_compressed_offsets.begin(),
                                                 d_compressed_offsets.begin() + 1,
                                                 d_decompressed_offsets.begin(),
                                                 d_decompressed_offsets.begin() + 1);
      auto span_it =
        thrust::make_zip_iterator(d_compressed_spans.begin(), d_decompressed_spans.begin());
      thrust::transform(
        rmm::exec_policy_nosync(stream),
        offset_it,
        offset_it + num_blocks(),
        span_it,
        bgzip_nvcomp_transform_functor{reinterpret_cast<uint8_t const*>(d_compressed_blocks.data()),
                                       reinterpret_cast<uint8_t*>(d_decompressed_blocks.begin())});
      if (decompressed_size() > 0) {
        if (nvcomp::is_decompression_disabled(nvcomp::compression_type::DEFLATE)) {
          gpuinflate(d_compressed_spans,
                     d_decompressed_spans,
                     d_decompression_results,
                     gzip_header_included::NO,
                     stream);
        } else {
          cudf::io::nvcomp::batched_decompress(cudf::io::nvcomp::compression_type::DEFLATE,
                                               d_compressed_spans,
                                               d_decompressed_spans,
                                               d_decompression_results,
                                               max_decompressed_size,
                                               decompressed_size(),
                                               stream);
        }
      }
      is_decompressed = true;
    }

    void reset()
    {
      h_compressed_blocks.resize(0);
      h_compressed_offsets.resize(1);
      h_decompressed_offsets.resize(1);
      // shrinking doesn't allocate/free, so we don't need to worry about streams
      auto stream = cudf::get_default_stream();
      d_compressed_blocks.resize(0, stream);
      d_decompressed_blocks.resize(0, stream);
      d_compressed_offsets.resize(0, stream);
      d_decompressed_offsets.resize(0, stream);
      d_compressed_spans.resize(0, stream);
      d_decompressed_spans.resize(0, stream);
      d_decompression_results.resize(0, stream);
      compressed_size_with_headers = 0;
      max_decompressed_size        = 0;
      available_decompressed_size  = 0;
      read_pos                     = 0;
      is_decompressed              = false;
    }

    [[nodiscard]] std::size_t num_blocks() const { return h_compressed_offsets.size() - 1; }

    [[nodiscard]] std::size_t compressed_size() const { return h_compressed_offsets.back(); }

    [[nodiscard]] std::size_t decompressed_size() const { return h_decompressed_offsets.back(); }

    [[nodiscard]] std::size_t remaining_size() const
    {
      return available_decompressed_size - read_pos;
    }

    void read_block(detail::bgzip::header header, std::istream& stream)
    {
      h_compressed_blocks.resize(h_compressed_blocks.size() + header.data_size());
      stream.read(h_compressed_blocks.data() + compressed_size(), header.data_size());
    }

    void add_block_offsets(detail::bgzip::header header, detail::bgzip::footer footer)
    {
      max_decompressed_size =
        std::max<std::size_t>(footer.decompressed_size, max_decompressed_size);
      h_compressed_offsets.push_back(compressed_size() + header.data_size());
      h_decompressed_offsets.push_back(decompressed_size() + footer.decompressed_size);
    }

    void consume_bytes(std::size_t size)
    {
      CUDF_EXPECTS(size <= remaining_size(), "out of bounds");
      read_pos += size;
    }
  };

  void read_next_compressed_chunk(std::size_t requested_size)
  {
    std::swap(_curr_blocks, _prev_blocks);
    if (_curr_blocks.is_decompressed) {
      // synchronize on the last decompression + copy, so we don't clobber any buffers
      CUDF_CUDA_TRY(hipEventSynchronize(_curr_blocks.event));
    }
    _curr_blocks.reset();
    // read chunks until we have enough decompressed data
    while (_curr_blocks.decompressed_size() < requested_size) {
      // calling peek on an already EOF stream causes it to fail, we need to avoid that
      if (_data_stream->eof()) { break; }
      // peek is necessary if we are already at the end, but didn't try to read another byte
      _data_stream->peek();
      if (_data_stream->eof() || _compressed_pos > _compressed_end) { break; }
      auto header = detail::bgzip::read_header(*_data_stream);
      _curr_blocks.read_block(header, *_data_stream);
      auto footer = detail::bgzip::read_footer(*_data_stream);
      _curr_blocks.add_block_offsets(header, footer);
      // for the last GZIP block, we restrict ourselves to the bytes up to _local_end
      // but only for the reader, not for decompression!
      if (_compressed_pos == _compressed_end) {
        _curr_blocks.available_decompressed_size += _local_end;
        _compressed_pos += header.block_size;
        break;
      } else {
        _curr_blocks.available_decompressed_size += footer.decompressed_size;
        _compressed_pos += header.block_size;
      }
    }
  }

  constexpr static std::size_t chunk_load_size = 1 << 24;  // load 16 MB of data by default

 public:
  bgzip_data_chunk_reader(std::unique_ptr<std::istream> input_stream,
                          uint64_t virtual_begin,
                          uint64_t virtual_end)
    : _data_stream(std::move(input_stream)),
      _prev_blocks{cudf::get_default_stream()},  // here we can use the default stream because
      _curr_blocks{cudf::get_default_stream()},  // we only initialize empty device_uvectors
      _local_end{virtual_end & 0xFFFFu},
      _compressed_pos{virtual_begin >> 16},
      _compressed_end{virtual_end >> 16}
  {
    // set failbit to throw on IO failures
    _data_stream->exceptions(std::istream::failbit);
    // seek to the beginning of the provided compressed offset
    _data_stream->seekg(_compressed_pos, std::ios_base::cur);
    // read the first blocks
    read_next_compressed_chunk(chunk_load_size);
    // seek to the beginning of the provided local offset
    auto const local_pos = virtual_begin & 0xFFFFu;
    if (local_pos > 0) {
      CUDF_EXPECTS(_curr_blocks.h_decompressed_offsets.size() > 1 &&
                     local_pos < _curr_blocks.h_decompressed_offsets[1],
                   "local part of virtual offset is out of bounds");
      _curr_blocks.consume_bytes(local_pos);
    }
  }

  void skip_bytes(std::size_t read_size) override
  {
    while (read_size > _curr_blocks.remaining_size()) {
      read_size -= _curr_blocks.remaining_size();
      _curr_blocks.consume_bytes(_curr_blocks.remaining_size());
      read_next_compressed_chunk(chunk_load_size);
      // calling peek on an already EOF stream causes it to fail, we need to avoid that
      if (_data_stream->eof()) { break; }
      // peek is necessary if we are already at the end, but didn't try to read another byte
      _data_stream->peek();
      if (_data_stream->eof() || _compressed_pos > _compressed_end) { break; }
    }
    read_size = std::min(read_size, _curr_blocks.remaining_size());
    _curr_blocks.consume_bytes(read_size);
  }

  std::unique_ptr<device_data_chunk> get_next_chunk(std::size_t read_size,
                                                    rmm::cuda_stream_view stream) override
  {
    CUDF_FUNC_RANGE();
    if (read_size <= _curr_blocks.remaining_size()) {
      _curr_blocks.decompress(stream);
      rmm::device_uvector<char> data(read_size, stream);
      CUDF_CUDA_TRY(
        hipMemcpyAsync(data.data(),
                        _curr_blocks.d_decompressed_blocks.data() + _curr_blocks.read_pos,
                        read_size,
                        hipMemcpyDeviceToDevice,
                        stream.value()));
      // record the host-to-device copy, decompression and device copy
      CUDF_CUDA_TRY(hipEventRecord(_curr_blocks.event, stream.value()));
      _curr_blocks.consume_bytes(read_size);
      return std::make_unique<device_uvector_data_chunk>(std::move(data));
    }
    read_next_compressed_chunk(read_size /* - _curr_blocks.remaining_size()*/);
    _prev_blocks.decompress(stream);
    _curr_blocks.decompress(stream);
    read_size = std::min(read_size, _prev_blocks.remaining_size() + _curr_blocks.remaining_size());
    rmm::device_uvector<char> data(read_size, stream);
    CUDF_CUDA_TRY(hipMemcpyAsync(data.data(),
                                  _prev_blocks.d_decompressed_blocks.data() + _prev_blocks.read_pos,
                                  _prev_blocks.remaining_size(),
                                  hipMemcpyDeviceToDevice,
                                  stream.value()));
    CUDF_CUDA_TRY(hipMemcpyAsync(data.data() + _prev_blocks.remaining_size(),
                                  _curr_blocks.d_decompressed_blocks.data() + _curr_blocks.read_pos,
                                  read_size - _prev_blocks.remaining_size(),
                                  hipMemcpyDeviceToDevice,
                                  stream.value()));
    // record the host-to-device copy, decompression and device copy
    CUDF_CUDA_TRY(hipEventRecord(_curr_blocks.event, stream.value()));
    CUDF_CUDA_TRY(hipEventRecord(_prev_blocks.event, stream.value()));
    read_size -= _prev_blocks.remaining_size();
    _prev_blocks.consume_bytes(_prev_blocks.remaining_size());
    _curr_blocks.consume_bytes(read_size);
    return std::make_unique<device_uvector_data_chunk>(std::move(data));
  }

 private:
  std::unique_ptr<std::istream> _data_stream;
  decompression_blocks _prev_blocks;
  decompression_blocks _curr_blocks;
  std::size_t _local_end;
  std::size_t _compressed_pos;
  std::size_t _compressed_end;
};

class bgzip_data_chunk_source : public data_chunk_source {
 public:
  bgzip_data_chunk_source(std::string_view filename, uint64_t virtual_begin, uint64_t virtual_end)
    : _filename{filename}, _virtual_begin{virtual_begin}, _virtual_end{virtual_end}
  {
  }

  [[nodiscard]] std::unique_ptr<data_chunk_reader> create_reader() const override
  {
    return std::make_unique<bgzip_data_chunk_reader>(
      std::make_unique<std::ifstream>(_filename, std::ifstream::in), _virtual_begin, _virtual_end);
  }

 private:
  std::string _filename;
  uint64_t _virtual_begin;
  uint64_t _virtual_end;
};

}  // namespace

std::unique_ptr<data_chunk_source> make_source_from_bgzip_file(std::string_view filename,
                                                               uint64_t virtual_begin,
                                                               uint64_t virtual_end)
{
  return std::make_unique<bgzip_data_chunk_source>(filename, virtual_begin, virtual_end);
}

std::unique_ptr<data_chunk_source> make_source_from_bgzip_file(std::string_view filename)
{
  return std::make_unique<bgzip_data_chunk_source>(
    filename, 0, std::numeric_limits<uint64_t>::max());
}

}  // namespace cudf::io::text
