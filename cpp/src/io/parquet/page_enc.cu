#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019-2020, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */
#include <hipcub/hipcub.hpp>
#include <io/parquet/parquet_gpu.hpp>
#include <io/utilities/block_utils.cuh>

#include <chrono>
#include <cudf/detail/utilities/cuda.cuh>

#include <thrust/gather.h>
#include <thrust/iterator/discard_iterator.h>

namespace cudf {
namespace io {
namespace parquet {
namespace gpu {
// Spark doesn't support RLE encoding for BOOLEANs
#ifdef ENABLE_BOOL_RLE
constexpr bool enable_bool_rle = true;
#else
constexpr bool enable_bool_rle = false;
#endif

#define INIT_HASH_BITS 12

struct frag_init_state_s {
  EncColumnDesc col;
  PageFragment frag;
  uint32_t total_dupes;
  size_type start_value_idx;
  volatile uint32_t scratch_red[32];
  uint32_t dict[MAX_PAGE_FRAGMENT_SIZE];
  union {
    uint16_t u16[1 << (INIT_HASH_BITS)];
    uint32_t u32[1 << (INIT_HASH_BITS - 1)];
  } map;
};

#define LOG2_RLE_BFRSZ 9
#define RLE_BFRSZ (1 << LOG2_RLE_BFRSZ)
#define RLE_MAX_LIT_RUN 0xfff8  // Maximum literal run for 2-byte run code

struct page_enc_state_s {
  uint8_t *cur;          //!< current output ptr
  uint8_t *rle_out;      //!< current RLE write ptr
  uint32_t rle_run;      //!< current RLE run
  uint32_t run_val;      //!< current RLE run value
  uint32_t rle_pos;      //!< RLE encoder positions
  uint32_t rle_numvals;  //!< RLE input value count
  uint32_t rle_lit_count;
  uint32_t rle_rpt_count;
  uint32_t page_start_val;
  volatile uint32_t rpt_map[4];
  volatile uint32_t scratch_red[32];
  EncPage page;
  EncColumnChunk ck;
  EncColumnDesc col;
  gpu_inflate_input_s comp_in;
  gpu_inflate_status_s comp_out;
  uint16_t vals[RLE_BFRSZ];
};

/**
 * @brief Return a 12-bit hash from a byte sequence
 */
inline __device__ uint32_t nvstr_init_hash(const uint8_t *ptr, uint32_t len)
{
  if (len != 0) {
    return (ptr[0] + (ptr[len - 1] << 5) + (len << 10)) & ((1 << INIT_HASH_BITS) - 1);
  } else {
    return 0;
  }
}

inline __device__ uint32_t uint32_init_hash(uint32_t v)
{
  return (v + (v >> 11) + (v >> 22)) & ((1 << INIT_HASH_BITS) - 1);
}

inline __device__ uint32_t uint64_init_hash(uint64_t v)
{
  return uint32_init_hash(static_cast<uint32_t>(v + (v >> 32)));
}

/**
 * @brief Initializes encoder page fragments
 *
 * Based on the number of rows in each fragment, populates the value count, the size of data in the
 * fragment, the number of unique values, and the data size of unique values.
 *
 * @param[in] frag Fragment array [fragment_id][column_id]
 * @param[in] col_desc Column description array [column_id]
 * @param[in] num_fragments Number of fragments per column
 * @param[in] num_columns Number of columns
 *
 **/
// blockDim {512,1,1}
template <int block_size>
__global__ void __launch_bounds__(block_size) gpuInitPageFragments(PageFragment *frag,
                                                                   const EncColumnDesc *col_desc,
                                                                   int32_t num_fragments,
                                                                   int32_t num_columns,
                                                                   uint32_t fragment_size,
                                                                   uint32_t max_num_rows)
{
  __shared__ __align__(16) frag_init_state_s state_g;

  using warp_reduce      = hipcub::WarpReduce<uint32_t>;
  using half_warp_reduce = hipcub::WarpReduce<uint32_t, 16>;
  __shared__ union {
    typename warp_reduce::TempStorage full[block_size / 32];
    typename half_warp_reduce::TempStorage half;
  } temp_storage;

  frag_init_state_s *const s = &state_g;
  uint32_t t                 = threadIdx.x;
  uint32_t start_row, dtype_len, dtype_len_in, dtype;

  if (t == 0) s->col = col_desc[blockIdx.x];
  for (uint32_t i = 0; i < sizeof(s->map) / sizeof(uint32_t); i += block_size) {
    if (i + t < sizeof(s->map) / sizeof(uint32_t)) s->map.u32[i + t] = 0;
  }
  __syncthreads();
  start_row = blockIdx.y * fragment_size;
  if (!t) {
    s->col.num_rows = min(s->col.num_rows, max_num_rows);
    // frag.num_rows = fragment_size except for the last page fragment which can be smaller.
    // num_rows is fixed but fragment size could be larger if the data is strings or nested.
    s->frag.num_rows           = min(fragment_size, max_num_rows - min(start_row, max_num_rows));
    s->frag.non_nulls          = 0;
    s->frag.num_dict_vals      = 0;
    s->frag.fragment_data_size = 0;
    s->frag.dict_data_size     = 0;
    s->total_dupes             = 0;

    // To use num_vals instead of num_rows, we need to calculate num_vals on the fly.
    // For list<list<int>>, values between i and i+50 can be calculated by
    // off_11 = off[i], off_12 = off[i+50]
    // off_21 = child.off[off_11], off_22 = child.off[off_12]
    // etc...
    s->start_value_idx      = start_row;
    size_type end_value_idx = start_row + s->frag.num_rows;
    for (size_type i = 0; i < s->col.nesting_levels; i++) {
      s->start_value_idx = s->col.nesting_offsets[i][s->start_value_idx];
      end_value_idx      = s->col.nesting_offsets[i][end_value_idx];
    }
    s->frag.start_value_idx = s->start_value_idx;
    s->frag.num_leaf_values = end_value_idx - s->start_value_idx;

    if (s->col.nesting_levels > 0) {
      // For nested schemas, the number of values in a fragment is not directly related to the
      // number of encoded data elements or the number of rows.  It is simply the number of
      // repetition/definition values which together encode validity and nesting information.
      size_type first_level_val_idx = s->col.level_offsets[start_row];
      size_type last_level_val_idx  = s->col.level_offsets[start_row + s->frag.num_rows];
      s->frag.num_values            = last_level_val_idx - first_level_val_idx;
    } else {
      s->frag.num_values = s->frag.num_rows;
    }
  }
  dtype = s->col.physical_type;
  dtype_len =
    (dtype == INT96) ? 12 : (dtype == INT64 || dtype == DOUBLE) ? 8 : (dtype == BOOLEAN) ? 1 : 4;
  if (dtype == INT32) {
    dtype_len_in = GetDtypeLogicalLen(s->col.converted_type);
  } else if (dtype == INT96) {
    // cudf doesn't support INT96 internally and uses INT64, so treat INT96 as an INT64 for
    // computing dictionary hash values and reading the data, but we do treat it as 12 bytes for
    // dtype_len, which determines how much memory we need to allocate for the fragment.
    dtype_len_in = 8;
  } else {
    dtype_len_in = (dtype == BYTE_ARRAY) ? sizeof(nvstrdesc_s) : dtype_len;
  }
  __syncthreads();

  size_type nvals           = s->frag.num_leaf_values;
  size_type start_value_idx = s->start_value_idx;

  for (uint32_t i = 0; i < nvals; i += block_size) {
    const uint32_t *valid = s->col.valid_map_base;
    uint32_t val_idx      = start_value_idx + i + t;
    uint32_t is_valid     = (i + t < nvals && val_idx < s->col.num_values)
                          ? (valid) ? (valid[val_idx >> 5] >> (val_idx & 0x1f)) & 1 : 1
                          : 0;
    uint32_t valid_warp = BALLOT(is_valid);
    uint32_t len, nz_pos, hash;
    if (is_valid) {
      len = dtype_len;
      if (dtype != BOOLEAN) {
        if (dtype == BYTE_ARRAY) {
          const char *ptr = static_cast<const nvstrdesc_s *>(s->col.column_data_base)[val_idx].ptr;
          uint32_t count =
            (uint32_t) reinterpret_cast<const nvstrdesc_s *>(s->col.column_data_base)[val_idx]
              .count;
          len += count;
          hash = nvstr_init_hash(reinterpret_cast<const uint8_t *>(ptr), count);
        } else if (dtype_len_in == 8) {
          hash = uint64_init_hash(static_cast<const uint64_t *>(s->col.column_data_base)[val_idx]);
        } else {
          hash = uint32_init_hash(
            (dtype_len_in == 4)
              ? static_cast<const uint32_t *>(s->col.column_data_base)[val_idx]
              : (dtype_len_in == 2)
                  ? static_cast<const uint16_t *>(s->col.column_data_base)[val_idx]
                  : static_cast<const uint8_t *>(s->col.column_data_base)[val_idx]);
        }
      }
    } else {
      len = 0;
    }

    nz_pos =
      s->frag.non_nulls + __popc(valid_warp & (0x7fffffffu >> (0x1fu - ((uint32_t)t & 0x1f))));
    len = warp_reduce(temp_storage.full[t / 32]).Sum(len);
    if (!(t & 0x1f)) {
      s->scratch_red[(t >> 5) + 0]  = __popc(valid_warp);
      s->scratch_red[(t >> 5) + 16] = len;
    }
    __syncthreads();
    if (t < 32) {
      uint32_t warp_pos  = WarpReducePos16((t < 16) ? s->scratch_red[t] : 0, t);
      uint32_t non_nulls = SHFL(warp_pos, 0xf);
      len = half_warp_reduce(temp_storage.half).Sum((t < 16) ? s->scratch_red[t + 16] : 0);
      if (t < 16) { s->scratch_red[t] = warp_pos; }
      if (!t) {
        s->frag.non_nulls = s->frag.non_nulls + non_nulls;
        s->frag.fragment_data_size += len;
      }
    }
    __syncthreads();
    if (is_valid && dtype != BOOLEAN) {
      uint32_t *dict_index = s->col.dict_index;
      if (t >= 32) { nz_pos += s->scratch_red[(t - 32) >> 5]; }
      if (dict_index) {
        atomicAdd(&s->map.u32[hash >> 1], (hash & 1) ? 1 << 16 : 1);
        dict_index[start_value_idx + nz_pos] =
          ((i + t) << INIT_HASH_BITS) |
          hash;  // Store the hash along with the index, so we don't have to recompute it
      }
    }
    __syncthreads();
  }
  __syncthreads();
  // Reorder the 16-bit local indices according to the hash values
  if (s->col.dict_index) {
#if (INIT_HASH_BITS != 12)
#error "Hardcoded for INIT_HASH_BITS=12"
#endif
    // Cumulative sum of hash map counts
    uint32_t count01 = s->map.u32[t * 4 + 0];
    uint32_t count23 = s->map.u32[t * 4 + 1];
    uint32_t count45 = s->map.u32[t * 4 + 2];
    uint32_t count67 = s->map.u32[t * 4 + 3];
    uint32_t sum01   = count01 + (count01 << 16);
    uint32_t sum23   = count23 + (count23 << 16);
    uint32_t sum45   = count45 + (count45 << 16);
    uint32_t sum67   = count67 + (count67 << 16);
    uint32_t sum_w, tmp;
    sum23 += (sum01 >> 16) * 0x10001;
    sum45 += (sum23 >> 16) * 0x10001;
    sum67 += (sum45 >> 16) * 0x10001;
    sum_w = sum67 >> 16;
    sum_w = WarpReducePos16(sum_w, t);
    if ((t & 0xf) == 0xf) { s->scratch_red[t >> 4] = sum_w; }
    __syncthreads();
    if (t < 32) {
      uint32_t sum_b    = WarpReducePos32(s->scratch_red[t], t);
      s->scratch_red[t] = sum_b;
    }
    __syncthreads();
    tmp                   = (t >= 16) ? s->scratch_red[(t >> 4) - 1] : 0;
    sum_w                 = (sum_w - (sum67 >> 16) + tmp) * 0x10001;
    s->map.u32[t * 4 + 0] = sum_w + sum01 - count01;
    s->map.u32[t * 4 + 1] = sum_w + sum23 - count23;
    s->map.u32[t * 4 + 2] = sum_w + sum45 - count45;
    s->map.u32[t * 4 + 3] = sum_w + sum67 - count67;
    __syncthreads();
  }
  // Put the indices back in hash order
  if (s->col.dict_index) {
    uint32_t *dict_index = s->col.dict_index + start_row;
    uint32_t nnz         = s->frag.non_nulls;
    for (uint32_t i = 0; i < nnz; i += block_size) {
      uint32_t pos = 0, hash = 0, pos_old, pos_new, sh, colliding_row, val = 0;
      bool collision;
      if (i + t < nnz) {
        val     = dict_index[i + t];
        hash    = val & ((1 << INIT_HASH_BITS) - 1);
        sh      = (hash & 1) ? 16 : 0;
        pos_old = s->map.u16[hash];
      }
      // The isolation of the atomicAdd, along with pos_old/pos_new is to guarantee deterministic
      // behavior for the first row in the hash map that will be used for early duplicate detection
      __syncthreads();
      if (i + t < nnz) {
        pos          = (atomicAdd(&s->map.u32[hash >> 1], 1 << sh) >> sh) & 0xffff;
        s->dict[pos] = val;
      }
      __syncthreads();
      collision = false;
      if (i + t < nnz) {
        pos_new   = s->map.u16[hash];
        collision = (pos != pos_old && pos_new > pos_old + 1);
        if (collision) { colliding_row = s->dict[pos_old]; }
      }
      __syncthreads();
      if (collision) { atomicMin(&s->dict[pos_old], val); }
      __syncthreads();
      // Resolve collision
      if (collision && val == s->dict[pos_old]) { s->dict[pos] = colliding_row; }
    }
    __syncthreads();
    // Now that the values are ordered by hash, compare every entry with the first entry in the hash
    // map, the position of the first entry can be inferred from the hash map counts
    uint32_t dupe_data_size = 0;
    for (uint32_t i = 0; i < nnz; i += block_size) {
      const void *col_data = s->col.column_data_base;
      uint32_t ck_row = 0, ck_row_ref = 0, is_dupe = 0, dupe_mask, dupes_before;
      if (i + t < nnz) {
        uint32_t dict_val = s->dict[i + t];
        uint32_t hash     = dict_val & ((1 << INIT_HASH_BITS) - 1);
        ck_row            = start_row + (dict_val >> INIT_HASH_BITS);
        ck_row_ref = start_row + (s->dict[(hash > 0) ? s->map.u16[hash - 1] : 0] >> INIT_HASH_BITS);
        if (ck_row_ref != ck_row) {
          if (dtype == BYTE_ARRAY) {
            const nvstrdesc_s *ck_data = static_cast<const nvstrdesc_s *>(col_data);
            const char *str1           = ck_data[ck_row].ptr;
            uint32_t len1              = (uint32_t)ck_data[ck_row].count;
            const char *str2           = ck_data[ck_row_ref].ptr;
            uint32_t len2              = (uint32_t)ck_data[ck_row_ref].count;
            is_dupe                    = nvstr_is_equal(str1, len1, str2, len2);
            dupe_data_size += (is_dupe) ? 4 + len1 : 0;
          } else {
            if (dtype_len_in == 8) {
              uint64_t v1 = static_cast<const uint64_t *>(col_data)[ck_row];
              uint64_t v2 = static_cast<const uint64_t *>(col_data)[ck_row_ref];
              is_dupe     = (v1 == v2);
              dupe_data_size += (is_dupe) ? 8 : 0;
            } else {
              uint32_t v1, v2;
              if (dtype_len_in == 4) {
                v1 = static_cast<const uint32_t *>(col_data)[ck_row];
                v2 = static_cast<const uint32_t *>(col_data)[ck_row_ref];
              } else if (dtype_len_in == 2) {
                v1 = static_cast<const uint16_t *>(col_data)[ck_row];
                v2 = static_cast<const uint16_t *>(col_data)[ck_row_ref];
              } else {
                v1 = static_cast<const uint8_t *>(col_data)[ck_row];
                v2 = static_cast<const uint8_t *>(col_data)[ck_row_ref];
              }
              is_dupe = (v1 == v2);
              dupe_data_size += (is_dupe) ? 4 : 0;
            }
          }
        }
      }
      dupe_mask    = BALLOT(is_dupe);
      dupes_before = s->total_dupes + __popc(dupe_mask & ((2 << (t & 0x1f)) - 1));
      if (!(t & 0x1f)) { s->scratch_red[t >> 5] = __popc(dupe_mask); }
      __syncthreads();
      if (t < 32) {
        uint32_t warp_dupes = (t < 16) ? s->scratch_red[t] : 0;
        uint32_t warp_pos   = WarpReducePos16(warp_dupes, t);
        if (t == 0xf) { s->total_dupes += warp_pos; }
        if (t < 16) { s->scratch_red[t] = warp_pos - warp_dupes; }
      }
      __syncthreads();
      if (i + t < nnz) {
        if (!is_dupe) {
          dupes_before += s->scratch_red[t >> 5];
          s->col.dict_data[start_row + i + t - dupes_before] = ck_row;
        } else {
          s->col.dict_index[ck_row] = ck_row_ref | (1u << 31);
        }
      }
    }
    __syncthreads();
    dupe_data_size = warp_reduce(temp_storage.full[t / 32]).Sum(dupe_data_size);
    if (!(t & 0x1f)) { s->scratch_red[t >> 5] = dupe_data_size; }
    __syncthreads();
    if (t < 32) {
      dupe_data_size = half_warp_reduce(temp_storage.half).Sum((t < 16) ? s->scratch_red[t] : 0);
      if (!t) {
        s->frag.dict_data_size = s->frag.fragment_data_size - dupe_data_size;
        s->frag.num_dict_vals  = s->frag.non_nulls - s->total_dupes;
      }
    }
  }
  __syncthreads();
  if (t == 0) frag[blockIdx.x * num_fragments + blockIdx.y] = s->frag;
}

// blockDim {128,1,1}
__global__ void __launch_bounds__(128) gpuInitFragmentStats(statistics_group *groups,
                                                            const PageFragment *fragments,
                                                            const EncColumnDesc *col_desc,
                                                            int32_t num_fragments,
                                                            int32_t num_columns,
                                                            uint32_t fragment_size)
{
  __shared__ __align__(8) statistics_group group_g[4];

  uint32_t lane_id          = threadIdx.x & 0x1f;
  uint32_t frag_id          = blockIdx.y * 4 + (threadIdx.x >> 5);
  uint32_t column_id        = blockIdx.x;
  statistics_group *const g = &group_g[threadIdx.x >> 5];
  if (!lane_id && frag_id < num_fragments) {
    g->col       = &col_desc[column_id];
    g->start_row = fragments[column_id * num_fragments + frag_id].start_value_idx;
    g->num_rows  = fragments[column_id * num_fragments + frag_id].num_leaf_values;
  }
  __syncthreads();
  if (frag_id < num_fragments and lane_id == 0) groups[column_id * num_fragments + frag_id] = *g;
}

// blockDim {128,1,1}
__global__ void __launch_bounds__(128) gpuInitPages(EncColumnChunk *chunks,
                                                    EncPage *pages,
                                                    const EncColumnDesc *col_desc,
                                                    statistics_merge_group *page_grstats,
                                                    statistics_merge_group *chunk_grstats,
                                                    int32_t num_rowgroups,
                                                    int32_t num_columns)
{
  __shared__ __align__(8) EncColumnDesc col_g;
  __shared__ __align__(8) EncColumnChunk ck_g;
  __shared__ __align__(8) PageFragment frag_g;
  __shared__ __align__(8) EncPage page_g;
  __shared__ __align__(8) statistics_merge_group pagestats_g;

  uint32_t t = threadIdx.x;

  if (t == 0) {
    col_g = col_desc[blockIdx.x];
    ck_g  = chunks[blockIdx.y * num_columns + blockIdx.x];
  }
  __syncthreads();
  if (t < 32) {
    uint32_t fragments_in_chunk  = 0;
    uint32_t rows_in_page        = 0;
    uint32_t values_in_page      = 0;
    uint32_t leaf_values_in_page = 0;
    uint32_t page_size           = 0;
    uint32_t num_pages           = 0;
    uint32_t num_rows            = 0;
    uint32_t page_start          = 0;
    uint32_t page_offset         = ck_g.ck_stat_size;
    uint32_t num_dict_entries    = 0;
    uint32_t comp_page_offset    = ck_g.ck_stat_size;
    uint32_t cur_row             = ck_g.start_row;
    uint32_t ck_max_stats_len    = 0;
    uint32_t max_stats_len       = 0;

    if (!t) {
      pagestats_g.col         = &col_desc[blockIdx.x];
      pagestats_g.start_chunk = ck_g.first_fragment;
      pagestats_g.num_chunks  = 0;
    }
    if (ck_g.has_dictionary) {
      if (!t) {
        page_g.page_data       = ck_g.uncompressed_bfr + page_offset;
        page_g.compressed_data = ck_g.compressed_bfr + comp_page_offset;
        page_g.num_fragments   = 0;
        page_g.page_type       = PageType::DICTIONARY_PAGE;
        page_g.dict_bits_plus1 = 0;
        page_g.chunk_id        = blockIdx.y * num_columns + blockIdx.x;
        page_g.hdr_size        = 0;
        page_g.max_hdr_size    = 32;
        page_g.max_data_size   = ck_g.dictionary_size;
        page_g.start_row       = cur_row;
        page_g.num_rows        = ck_g.total_dict_entries;
        page_g.num_leaf_values = ck_g.total_dict_entries;
        page_g.num_values      = ck_g.total_dict_entries;
        page_offset += page_g.max_hdr_size + page_g.max_data_size;
        comp_page_offset += page_g.max_hdr_size + GetMaxCompressedBfrSize(page_g.max_data_size);
      }
      SYNCWARP();
      if (t == 0) {
        if (pages) pages[ck_g.first_page] = page_g;
        if (page_grstats) page_grstats[ck_g.first_page] = pagestats_g;
      }
      num_pages = 1;
    }
    SYNCWARP();
    // This loop goes over one page fragment at a time and adds it to page.
    // When page size crosses a particular limit, then it moves on to the next page and then next
    // page fragment gets added to that one.

    // This doesn't actually deal with data. It's agnostic. It only cares about number of rows and
    // page size.
    do {
      uint32_t fragment_data_size, max_page_size, minmax_len = 0;
      SYNCWARP();
      if (num_rows < ck_g.num_rows) {
        if (t == 0) { frag_g = ck_g.fragments[fragments_in_chunk]; }
        if (!t && ck_g.stats && col_g.stats_dtype == dtype_string) {
          minmax_len = max(ck_g.stats[fragments_in_chunk].min_value.str_val.length,
                           ck_g.stats[fragments_in_chunk].max_value.str_val.length);
        }
      } else if (!t) {
        frag_g.fragment_data_size = 0;
        frag_g.num_rows           = 0;
      }
      SYNCWARP();
      if (ck_g.has_dictionary && fragments_in_chunk < ck_g.num_dict_fragments) {
        fragment_data_size =
          frag_g.num_leaf_values * 2;  // Assume worst-case of 2-bytes per dictionary index
      } else {
        fragment_data_size = frag_g.fragment_data_size;
      }
      // TODO (dm): this convoluted logic to limit page size needs refactoring
      max_page_size = (values_in_page * 2 >= ck_g.num_values)
                        ? 256 * 1024
                        : (values_in_page * 3 >= ck_g.num_values) ? 384 * 1024 : 512 * 1024;
      if (num_rows >= ck_g.num_rows ||
          (values_in_page > 0 &&
           (page_size + fragment_data_size > max_page_size ||
            (ck_g.has_dictionary && fragments_in_chunk == ck_g.num_dict_fragments)))) {
        uint32_t dict_bits_plus1;

        if (ck_g.has_dictionary && page_start < ck_g.num_dict_fragments) {
          uint32_t dict_bits;
          if (num_dict_entries <= 2) {
            dict_bits = 1;
          } else if (num_dict_entries <= 4) {
            dict_bits = 2;
          } else if (num_dict_entries <= 16) {
            dict_bits = 4;
          } else if (num_dict_entries <= 256) {
            dict_bits = 8;
          } else if (num_dict_entries <= 4096) {
            dict_bits = 12;
          } else {
            dict_bits = 16;
          }
          page_size       = 1 + 5 + ((values_in_page * dict_bits + 7) >> 3) + (values_in_page >> 8);
          dict_bits_plus1 = dict_bits + 1;
        } else {
          dict_bits_plus1 = 0;
        }
        if (!t) {
          page_g.num_fragments   = fragments_in_chunk - page_start;
          page_g.chunk_id        = blockIdx.y * num_columns + blockIdx.x;
          page_g.page_type       = PageType::DATA_PAGE;
          page_g.dict_bits_plus1 = dict_bits_plus1;
          page_g.hdr_size        = 0;
          page_g.max_hdr_size    = 32;  // Max size excluding statistics
          if (ck_g.stats) {
            uint32_t stats_hdr_len = 16;
            if (col_g.stats_dtype == dtype_string) {
              stats_hdr_len += 5 * 3 + 2 * max_stats_len;
            } else {
              stats_hdr_len += ((col_g.stats_dtype >= dtype_int64) ? 10 : 5) * 3;
            }
            page_g.max_hdr_size += stats_hdr_len;
          }
          page_g.page_data        = ck_g.uncompressed_bfr + page_offset;
          page_g.compressed_data  = ck_g.compressed_bfr + comp_page_offset;
          page_g.start_row        = cur_row;
          page_g.num_rows         = rows_in_page;
          page_g.num_leaf_values  = leaf_values_in_page;
          page_g.num_values       = values_in_page;
          uint32_t def_level_bits = col_g.level_bits & 0xf;
          uint32_t rep_level_bits = col_g.level_bits >> 4;
          // Run length = 4, max(rle/bitpack header) = 5, add one byte per 256 values for overhead
          // TODO (dm): Improve readability of these calculations.
          uint32_t def_level_size =
            (def_level_bits != 0)
              ? 4 + 5 + ((def_level_bits * page_g.num_values + 7) >> 3) + (page_g.num_values >> 8)
              : 0;
          uint32_t rep_level_size =
            (rep_level_bits != 0)
              ? 4 + 5 + ((rep_level_bits * page_g.num_values + 7) >> 3) + (page_g.num_values >> 8)
              : 0;
          page_g.max_data_size = page_size + def_level_size + rep_level_size;

          pagestats_g.start_chunk = ck_g.first_fragment + page_start;
          pagestats_g.num_chunks  = page_g.num_fragments;
          page_offset += page_g.max_hdr_size + page_g.max_data_size;
          comp_page_offset += page_g.max_hdr_size + GetMaxCompressedBfrSize(page_g.max_data_size);
          cur_row += rows_in_page;
          ck_max_stats_len = max(ck_max_stats_len, max_stats_len);
        }
        SYNCWARP();
        if (t == 0) {
          if (pages) { pages[ck_g.first_page + num_pages] = page_g; }

          if (page_grstats) { page_grstats[ck_g.first_page + num_pages] = pagestats_g; }
        }

        num_pages++;
        page_size           = 0;
        rows_in_page        = 0;
        values_in_page      = 0;
        leaf_values_in_page = 0;
        page_start          = fragments_in_chunk;
        max_stats_len       = 0;
      }
      max_stats_len = max(max_stats_len, minmax_len);
      num_dict_entries += frag_g.num_dict_vals;
      page_size += fragment_data_size;
      rows_in_page += frag_g.num_rows;
      values_in_page += frag_g.num_values;
      leaf_values_in_page += frag_g.num_leaf_values;
      num_rows += frag_g.num_rows;
      fragments_in_chunk++;
    } while (frag_g.num_rows != 0);
    SYNCWARP();
    if (!t) {
      if (ck_g.ck_stat_size == 0 && ck_g.stats) {
        uint32_t ck_stat_size = 48 + 2 * ck_max_stats_len;
        page_offset += ck_stat_size;
        comp_page_offset += ck_stat_size;
        ck_g.ck_stat_size = ck_stat_size;
      }
      ck_g.num_pages          = num_pages;
      ck_g.bfr_size           = page_offset;
      ck_g.compressed_size    = comp_page_offset;
      pagestats_g.start_chunk = ck_g.first_page + ck_g.has_dictionary;  // Exclude dictionary
      pagestats_g.num_chunks  = num_pages - ck_g.has_dictionary;
    }
  }
  __syncthreads();
  if (t == 0) {
    chunks[blockIdx.y * num_columns + blockIdx.x] = ck_g;
    if (chunk_grstats) chunk_grstats[blockIdx.y * num_columns + blockIdx.x] = pagestats_g;
  }
}

/**
 * @brief Mask table representing how many consecutive repeats are needed to code a repeat run
 *[nbits-1]
 **/
static __device__ __constant__ uint32_t kRleRunMask[16] = {
  0x00ffffff, 0x0fff, 0x00ff, 0x3f, 0x0f, 0x0f, 0x7, 0x7, 0x3, 0x3, 0x3, 0x3, 0x1, 0x1, 0x1, 0x1};

/**
 * @brief Variable-length encode an integer
 **/
inline __device__ uint8_t *VlqEncode(uint8_t *p, uint32_t v)
{
  while (v > 0x7f) {
    *p++ = (v | 0x80);
    v >>= 7;
  }
  *p++ = v;
  return p;
}

/**
 * @brief Pack literal values in output bitstream (1,2,4,8,12 or 16 bits per value)
 **/
inline __device__ void PackLiterals(
  uint8_t *dst, uint32_t v, uint32_t count, uint32_t w, uint32_t t)
{
  if (w == 1 || w == 2 || w == 4 || w == 8 || w == 12 || w == 16) {
    if (t <= (count | 0x1f)) {
      if (w == 1 || w == 2 || w == 4) {
        uint32_t mask = 0;
        if (w == 1) {
          v |= SHFL_XOR(v, 1) << 1;
          v |= SHFL_XOR(v, 2) << 2;
          v |= SHFL_XOR(v, 4) << 4;
          mask = 0x7;
        } else if (w == 2) {
          v |= SHFL_XOR(v, 1) << 2;
          v |= SHFL_XOR(v, 2) << 4;
          mask = 0x3;
        } else if (w == 4) {
          v |= SHFL_XOR(v, 1) << 4;
          mask = 0x1;
        }
        if (t < count && mask && !(t & mask)) { dst[(t * w) >> 3] = v; }
        return;
      } else if (w == 8) {
        if (t < count) { dst[t] = v; }
        return;
      } else if (w == 12) {
        v |= SHFL_XOR(v, 1) << 12;
        if (t < count && !(t & 1)) {
          dst[(t >> 1) * 3 + 0] = v;
          dst[(t >> 1) * 3 + 1] = v >> 8;
          dst[(t >> 1) * 3 + 2] = v >> 16;
        }
        return;
      } else if (w == 16) {
        if (t < count) {
          dst[t * 2 + 0] = v;
          dst[t * 2 + 1] = v >> 8;
        }
        return;
      }
    } else {
      return;
    }
  } else {
    // Scratch space to temporarily write to. Needed because we will use atomics to write 32 bit
    // words but the destination mem may not be a multiple of 4 bytes.
    // TODO (dm): This assumes blockdim = 128 and max bits per value = 16. Reduce magic numbers.
    __shared__ uint32_t scratch[64];
    if (t < 64) { scratch[t] = 0; }
    __syncthreads();

    if (t <= count) {
      uint64_t v64 = v;
      v64 <<= (t * w) & 0x1f;

      // Copy 64 bit word into two 32 bit words while following C++ strict aliasing rules.
      uint32_t v32[2];
      memcpy(&v32, &v64, sizeof(uint64_t));

      // Atomically write result to scratch
      if (v32[0]) { atomicOr(scratch + ((t * w) >> 5), v32[0]); }
      if (v32[1]) { atomicOr(scratch + ((t * w) >> 5) + 1, v32[1]); }
    }
    __syncthreads();

    // Copy scratch data to final destination
    auto available_bytes = (count * w + 7) / 8;

    auto scratch_bytes = reinterpret_cast<char *>(&scratch[0]);
    if (t < available_bytes) { dst[t] = scratch_bytes[t]; }
    if (t + 128 < available_bytes) { dst[t + 128] = scratch_bytes[t + 128]; }
    __syncthreads();
  }
}

/**
 * @brief RLE encoder
 *
 * @param[in,out] s Page encode state
 * @param[in] numvals Total count of input values
 * @param[in] nbits number of bits per symbol (1..16)
 * @param[in] flush nonzero if last batch in block
 * @param[in] t thread id (0..127)
 */
static __device__ void RleEncode(
  page_enc_state_s *s, uint32_t numvals, uint32_t nbits, uint32_t flush, uint32_t t)
{
  uint32_t rle_pos = s->rle_pos;
  uint32_t rle_run = s->rle_run;

  while (rle_pos < numvals || (flush && rle_run)) {
    uint32_t pos = rle_pos + t;
    if (rle_run > 0 && !(rle_run & 1)) {
      // Currently in a long repeat run
      uint32_t mask = BALLOT(pos < numvals && s->vals[pos & (RLE_BFRSZ - 1)] == s->run_val);
      uint32_t rle_rpt_count, max_rpt_count;
      if (!(t & 0x1f)) { s->rpt_map[t >> 5] = mask; }
      __syncthreads();
      if (t < 32) {
        uint32_t c32 = BALLOT(t >= 4 || s->rpt_map[t] != 0xffffffffu);
        if (!t) {
          uint32_t last_idx = __ffs(c32) - 1;
          s->rle_rpt_count =
            last_idx * 32 + ((last_idx < 4) ? __ffs(~s->rpt_map[last_idx]) - 1 : 0);
        }
      }
      __syncthreads();
      max_rpt_count = min(numvals - rle_pos, 128);
      rle_rpt_count = s->rle_rpt_count;
      rle_run += rle_rpt_count << 1;
      rle_pos += rle_rpt_count;
      if (rle_rpt_count < max_rpt_count || (flush && rle_pos == numvals)) {
        if (t == 0) {
          uint32_t const run_val = s->run_val;
          uint8_t *dst           = VlqEncode(s->rle_out, rle_run);
          *dst++                 = run_val;
          if (nbits > 8) { *dst++ = run_val >> 8; }
          s->rle_out = dst;
        }
        rle_run = 0;
      }
    } else {
      // New run or in a literal run
      uint32_t v0      = s->vals[pos & (RLE_BFRSZ - 1)];
      uint32_t v1      = s->vals[(pos + 1) & (RLE_BFRSZ - 1)];
      uint32_t mask    = BALLOT(pos + 1 < numvals && v0 == v1);
      uint32_t maxvals = min(numvals - rle_pos, 128);
      uint32_t rle_lit_count, rle_rpt_count;
      if (!(t & 0x1f)) { s->rpt_map[t >> 5] = mask; }
      __syncthreads();
      if (t < 32) {
        // Repeat run can only start on a multiple of 8 values
        uint32_t idx8        = (t * 8) >> 5;
        uint32_t pos8        = (t * 8) & 0x1f;
        uint32_t m0          = (idx8 < 4) ? s->rpt_map[idx8] : 0;
        uint32_t m1          = (idx8 < 3) ? s->rpt_map[idx8 + 1] : 0;
        uint32_t needed_mask = kRleRunMask[nbits - 1];
        mask                 = BALLOT((__funnelshift_r(m0, m1, pos8) & needed_mask) == needed_mask);
        if (!t) {
          uint32_t rle_run_start = (mask != 0) ? min((__ffs(mask) - 1) * 8, maxvals) : maxvals;
          uint32_t rpt_len       = 0;
          if (rle_run_start < maxvals) {
            uint32_t idx_cur = rle_run_start >> 5;
            uint32_t idx_ofs = rle_run_start & 0x1f;
            while (idx_cur < 4) {
              m0   = (idx_cur < 4) ? s->rpt_map[idx_cur] : 0;
              m1   = (idx_cur < 3) ? s->rpt_map[idx_cur + 1] : 0;
              mask = ~__funnelshift_r(m0, m1, idx_ofs);
              if (mask != 0) {
                rpt_len += __ffs(mask) - 1;
                break;
              }
              rpt_len += 32;
              idx_cur++;
            }
          }
          s->rle_lit_count = rle_run_start;
          s->rle_rpt_count = min(rpt_len, maxvals - rle_run_start);
        }
      }
      __syncthreads();
      rle_lit_count = s->rle_lit_count;
      rle_rpt_count = s->rle_rpt_count;
      if (rle_lit_count != 0 || (rle_run != 0 && rle_rpt_count != 0)) {
        uint32_t lit_div8;
        bool need_more_data = false;
        if (!flush && rle_pos + rle_lit_count == numvals) {
          // Wait for more data
          rle_lit_count -= min(rle_lit_count, 24);
          need_more_data = true;
        }
        if (rle_lit_count != 0) {
          lit_div8 = (rle_lit_count + ((flush && rle_pos + rle_lit_count == numvals) ? 7 : 0)) >> 3;
          if (rle_run + lit_div8 * 2 > 0x7f) {
            lit_div8      = 0x3f - (rle_run >> 1);  // Limit to fixed 1-byte header (504 literals)
            rle_rpt_count = 0;                      // Defer repeat run
          }
          if (lit_div8 != 0) {
            uint8_t *dst = s->rle_out + 1 + (rle_run >> 1) * nbits;
            PackLiterals(dst, (rle_pos + t < numvals) ? v0 : 0, lit_div8 * 8, nbits, t);
            rle_run = (rle_run + lit_div8 * 2) | 1;
            rle_pos = min(rle_pos + lit_div8 * 8, numvals);
          }
        }
        if (rle_run >= ((rle_rpt_count != 0 || (flush && rle_pos == numvals)) ? 0x03 : 0x7f)) {
          __syncthreads();
          // Complete literal run
          if (!t) {
            uint8_t *dst = s->rle_out;
            dst[0]       = rle_run;  // At most 0x7f
            dst += 1 + nbits * (rle_run >> 1);
            s->rle_out = dst;
          }
          rle_run = 0;
        }
        if (need_more_data) { break; }
      }
      // Start a repeat run
      if (rle_rpt_count != 0) {
        if (t == s->rle_lit_count) { s->run_val = v0; }
        rle_run = rle_rpt_count * 2;
        rle_pos += rle_rpt_count;
        if (rle_pos + 1 == numvals && !flush) { break; }
      }
    }
    __syncthreads();
  }
  __syncthreads();
  if (!t) {
    s->rle_run     = rle_run;
    s->rle_pos     = rle_pos;
    s->rle_numvals = numvals;
  }
}

/**
 * @brief PLAIN bool encoder
 *
 * @param[in,out] s Page encode state
 * @param[in] numvals Total count of input values
 * @param[in] flush nonzero if last batch in block
 * @param[in] t thread id (0..127)
 */
static __device__ void PlainBoolEncode(page_enc_state_s *s,
                                       uint32_t numvals,
                                       uint32_t flush,
                                       uint32_t t)
{
  uint32_t rle_pos = s->rle_pos;
  uint8_t *dst     = s->rle_out;

  while (rle_pos < numvals) {
    uint32_t pos    = rle_pos + t;
    uint32_t v      = (pos < numvals) ? s->vals[pos & (RLE_BFRSZ - 1)] : 0;
    uint32_t n      = min(numvals - rle_pos, 128);
    uint32_t nbytes = (n + ((flush) ? 7 : 0)) >> 3;
    if (!nbytes) { break; }
    v |= SHFL_XOR(v, 1) << 1;
    v |= SHFL_XOR(v, 2) << 2;
    v |= SHFL_XOR(v, 4) << 4;
    if (t < n && !(t & 7)) { dst[t >> 3] = v; }
    rle_pos = min(rle_pos + nbytes * 8, numvals);
    dst += nbytes;
  }
  __syncthreads();
  if (!t) {
    s->rle_pos     = rle_pos;
    s->rle_numvals = numvals;
    s->rle_out     = dst;
  }
}

constexpr auto julian_calendar_epoch_diff()
{
  using namespace simt::std::chrono;
  using namespace simt::std::chrono_literals;
  return sys_days{January / 1 / 1970} - (sys_days{November / 24 / -4713} + 12h);
}

/**
 * @brief Converts a sys_time<nanoseconds> into a pair with nanoseconds since midnight and number of
 * Julian days. Does not deal with time zones. Used by INT96 code.
 *
 * @param ns number of nanoseconds since epoch
 * @return std::pair<nanoseconds,days> where nanoseconds is the number of nanoseconds
 * elapsed in the day and days is the number of days from Julian epoch.
 */
static __device__ std::pair<simt::std::chrono::nanoseconds, simt::std::chrono::days>
convert_nanoseconds(simt::std::chrono::sys_time<simt::std::chrono::nanoseconds> const ns)
{
  using namespace simt::std::chrono;
  auto const nanosecond_ticks = ns.time_since_epoch();
  auto const gregorian_days   = floor<days>(nanosecond_ticks);
  auto const julian_days      = gregorian_days + ceil<days>(julian_calendar_epoch_diff());

  auto const last_day_ticks = nanosecond_ticks - duration_cast<nanoseconds>(gregorian_days);
  return {last_day_ticks, julian_days};
}

// blockDim(128, 1, 1)
__global__ void __launch_bounds__(128, 8) gpuEncodePages(EncPage *pages,
                                                         const EncColumnChunk *chunks,
                                                         gpu_inflate_input_s *comp_in,
                                                         gpu_inflate_status_s *comp_out,
                                                         uint32_t start_page)
{
  __shared__ __align__(8) page_enc_state_s state_g;

  page_enc_state_s *const s = &state_g;
  uint32_t t                = threadIdx.x;
  uint32_t dtype, dtype_len_in, dtype_len_out;
  int32_t dict_bits;

  if (t == 0) {
    s->page = pages[start_page + blockIdx.x];
    s->ck   = chunks[s->page.chunk_id];
    s->col  = *s->ck.col_desc;
    s->cur  = s->page.page_data + s->page.max_hdr_size;
  }
  __syncthreads();

  // Encode Repetition and Definition levels
  if (s->page.page_type != PageType::DICTIONARY_PAGE && s->col.level_bits != 0 &&
      s->col.nesting_levels == 0) {
    // Calculate definition levels from validity
    const uint32_t *valid = s->col.valid_map_base;
    uint32_t def_lvl_bits = s->col.level_bits & 0xf;
    if (def_lvl_bits != 0) {
      if (!t) {
        s->rle_run     = 0;
        s->rle_pos     = 0;
        s->rle_numvals = 0;
        s->rle_out     = s->cur + 4;
      }
      __syncthreads();
      while (s->rle_numvals < s->page.num_rows) {
        uint32_t rle_numvals = s->rle_numvals;
        uint32_t nrows       = min(s->page.num_rows - rle_numvals, 128);
        uint32_t row         = s->page.start_row + rle_numvals + t;
        // Definition level encodes validity. Checks the valid map and if it is valid, then sets the
        // def_lvl accordingly and sets it in s->vals which is then given to RleEncode to encode
        uint32_t def_lvl = (rle_numvals + t < s->page.num_rows && row < s->col.num_rows)
                             ? (valid) ? (valid[row >> 5] >> (row & 0x1f)) & 1 : 1
                             : 0;
        s->vals[(rle_numvals + t) & (RLE_BFRSZ - 1)] = def_lvl;
        __syncthreads();
        rle_numvals += nrows;
        RleEncode(s, rle_numvals, def_lvl_bits, (rle_numvals == s->page.num_rows), t);
        __syncthreads();
      }
      if (t < 32) {
        uint8_t *cur     = s->cur;
        uint8_t *rle_out = s->rle_out;
        if (t < 4) {
          uint32_t rle_bytes = (uint32_t)(rle_out - cur) - 4;
          cur[t]             = rle_bytes >> (t * 8);
        }
        SYNCWARP();
        if (t == 0) { s->cur = rle_out; }
      }
    }
  } else if (s->page.page_type != PageType::DICTIONARY_PAGE && s->col.nesting_levels > 0) {
    auto encode_levels = [&](uint8_t const *lvl_val_data, uint32_t nbits) {
      // For list types, the repetition and definition levels are pre-calculated. We just need to
      // encode and write them now.
      if (!t) {
        s->rle_run     = 0;
        s->rle_pos     = 0;
        s->rle_numvals = 0;
        s->rle_out     = s->cur + 4;
      }
      __syncthreads();
      size_type page_first_val_idx = s->col.level_offsets[s->page.start_row];
      size_type col_last_val_idx   = s->col.level_offsets[s->col.num_rows];
      while (s->rle_numvals < s->page.num_values) {
        uint32_t rle_numvals = s->rle_numvals;
        uint32_t nvals       = min(s->page.num_values - rle_numvals, 128);
        uint32_t idx         = page_first_val_idx + rle_numvals + t;
        uint32_t lvl_val =
          (rle_numvals + t < s->page.num_values && idx < col_last_val_idx) ? lvl_val_data[idx] : 0;
        s->vals[(rle_numvals + t) & (RLE_BFRSZ - 1)] = lvl_val;
        __syncthreads();
        rle_numvals += nvals;
        RleEncode(s, rle_numvals, nbits, (rle_numvals == s->page.num_values), t);
        __syncthreads();
      }
      if (t < 32) {
        uint8_t *cur     = s->cur;
        uint8_t *rle_out = s->rle_out;
        if (t < 4) {
          uint32_t rle_bytes = (uint32_t)(rle_out - cur) - 4;
          cur[t]             = rle_bytes >> (t * 8);
        }
        SYNCWARP();
        if (t == 0) { s->cur = rle_out; }
      }
    };
    encode_levels(s->col.rep_values, s->col.level_bits >> 4);
    __syncthreads();
    encode_levels(s->col.def_values, s->col.level_bits & 0xf);
  }
  // Encode data values
  __syncthreads();
  dtype = s->col.physical_type;
  dtype_len_out =
    (dtype == INT96) ? 12 : (dtype == INT64 || dtype == DOUBLE) ? 8 : (dtype == BOOLEAN) ? 1 : 4;
  if (dtype == INT32) {
    dtype_len_in = GetDtypeLogicalLen(s->col.converted_type);
  } else if (dtype == INT96) {
    dtype_len_in = 8;
  } else {
    dtype_len_in = (dtype == BYTE_ARRAY) ? sizeof(nvstrdesc_s) : dtype_len_out;
  }
  dict_bits = (dtype == BOOLEAN) ? 1 : (s->page.dict_bits_plus1 - 1);
  if (t == 0) {
    uint8_t *dst   = s->cur;
    s->rle_run     = 0;
    s->rle_pos     = 0;
    s->rle_numvals = 0;
    s->rle_out     = dst;
    if (dict_bits >= 0 && dtype != BOOLEAN) {
      dst[0]     = dict_bits;
      s->rle_out = dst + 1;
    }
    s->page_start_val = s->page.start_row;
    for (size_type i = 0; i < s->col.nesting_levels; i++) {
      s->page_start_val = s->col.nesting_offsets[i][s->page_start_val];
    }
  }
  __syncthreads();
  for (uint32_t cur_val_idx = 0; cur_val_idx < s->page.num_leaf_values;) {
    uint32_t nvals   = min(s->page.num_leaf_values - cur_val_idx, 128);
    uint32_t val_idx = s->page_start_val + cur_val_idx + t;
    uint32_t is_valid, warp_valids, len, pos;

    if (s->page.page_type == PageType::DICTIONARY_PAGE) {
      is_valid = (cur_val_idx + t < s->page.num_leaf_values);
      val_idx  = (is_valid) ? s->col.dict_data[val_idx] : val_idx;
    } else {
      const uint32_t *valid = s->col.valid_map_base;
      is_valid = (val_idx < s->col.num_values && cur_val_idx + t < s->page.num_leaf_values)
                   ? (valid) ? (valid[val_idx >> 5] >> (val_idx & 0x1f)) & 1 : 1
                   : 0;
    }
    warp_valids = BALLOT(is_valid);
    cur_val_idx += nvals;
    if (dict_bits >= 0) {
      // Dictionary encoding
      if (dict_bits > 0) {
        uint32_t rle_numvals;

        pos = __popc(warp_valids & ((1 << (t & 0x1f)) - 1));
        if (!(t & 0x1f)) { s->scratch_red[t >> 5] = __popc(warp_valids); }
        __syncthreads();
        if (t < 32) { s->scratch_red[t] = WarpReducePos4((t < 4) ? s->scratch_red[t] : 0, t); }
        __syncthreads();
        pos         = pos + ((t >= 32) ? s->scratch_red[(t - 32) >> 5] : 0);
        rle_numvals = s->rle_numvals;
        if (is_valid) {
          uint32_t v;
          if (dtype == BOOLEAN) {
            v = reinterpret_cast<const uint8_t *>(s->col.column_data_base)[val_idx];
          } else {
            v = s->col.dict_index[val_idx];
          }
          s->vals[(rle_numvals + pos) & (RLE_BFRSZ - 1)] = v;
        }
        rle_numvals += s->scratch_red[3];
        __syncthreads();
        if ((!enable_bool_rle) && (dtype == BOOLEAN)) {
          PlainBoolEncode(s, rle_numvals, (cur_val_idx == s->page.num_leaf_values), t);
        } else {
          RleEncode(s, rle_numvals, dict_bits, (cur_val_idx == s->page.num_leaf_values), t);
        }
        __syncthreads();
      }
      if (t == 0) { s->cur = s->rle_out; }
      __syncthreads();
    } else {
      // Non-dictionary encoding
      uint8_t *dst = s->cur;

      if (is_valid) {
        len = dtype_len_out;
        if (dtype == BYTE_ARRAY) {
          len += (uint32_t) reinterpret_cast<const nvstrdesc_s *>(s->col.column_data_base)[val_idx]
                   .count;
        }
      } else {
        len = 0;
      }
      pos = WarpReducePos32(len, t);
      if ((t & 0x1f) == 0x1f) { s->scratch_red[t >> 5] = pos; }
      __syncthreads();
      if (t < 32) { s->scratch_red[t] = WarpReducePos4((t < 4) ? s->scratch_red[t] : 0, t); }
      __syncthreads();
      if (t == 0) { s->cur = dst + s->scratch_red[3]; }
      pos = pos + ((t >= 32) ? s->scratch_red[(t - 32) >> 5] : 0) - len;
      if (is_valid) {
        const uint8_t *src8 = reinterpret_cast<const uint8_t *>(s->col.column_data_base) +
                              val_idx * (size_t)dtype_len_in;
        switch (dtype) {
          case INT32:
          case FLOAT: {
            int32_t v;
            if (dtype_len_in == 4)
              v = *reinterpret_cast<const int32_t *>(src8);
            else if (dtype_len_in == 2)
              v = *reinterpret_cast<const int16_t *>(src8);
            else
              v = *reinterpret_cast<const int8_t *>(src8);
            dst[pos + 0] = v;
            dst[pos + 1] = v >> 8;
            dst[pos + 2] = v >> 16;
            dst[pos + 3] = v >> 24;
          } break;
          case INT64: {
            int64_t v        = *reinterpret_cast<const int64_t *>(src8);
            int32_t ts_scale = s->col.ts_scale;
            if (ts_scale != 0) {
              if (ts_scale < 0) {
                v /= -ts_scale;
              } else {
                v *= ts_scale;
              }
            }
            dst[pos + 0] = v;
            dst[pos + 1] = v >> 8;
            dst[pos + 2] = v >> 16;
            dst[pos + 3] = v >> 24;
            dst[pos + 4] = v >> 32;
            dst[pos + 5] = v >> 40;
            dst[pos + 6] = v >> 48;
            dst[pos + 7] = v >> 56;
          } break;
          case INT96: {
            int64_t v        = *reinterpret_cast<const int64_t *>(src8);
            int32_t ts_scale = s->col.ts_scale;
            if (ts_scale != 0) {
              if (ts_scale < 0) {
                v /= -ts_scale;
              } else {
                v *= ts_scale;
              }
            }

            auto const ret = convert_nanoseconds([&]() {
              using namespace simt::std::chrono;

              switch (s->col.converted_type) {
                case TIMESTAMP_MILLIS: {
                  return sys_time<nanoseconds>{milliseconds{v}};
                } break;
                case TIMESTAMP_MICROS: {
                  return sys_time<nanoseconds>{microseconds{v}};
                } break;
              }
              return sys_time<nanoseconds>{microseconds{0}};
            }());

            // the 12 bytes of fixed length data.
            v             = ret.first.count();
            dst[pos + 0]  = v;
            dst[pos + 1]  = v >> 8;
            dst[pos + 2]  = v >> 16;
            dst[pos + 3]  = v >> 24;
            dst[pos + 4]  = v >> 32;
            dst[pos + 5]  = v >> 40;
            dst[pos + 6]  = v >> 48;
            dst[pos + 7]  = v >> 56;
            uint32_t w    = ret.second.count();
            dst[pos + 8]  = w;
            dst[pos + 9]  = w >> 8;
            dst[pos + 10] = w >> 16;
            dst[pos + 11] = w >> 24;
          } break;

          case DOUBLE: memcpy(dst + pos, src8, 8); break;
          case BYTE_ARRAY: {
            const char *str_data = reinterpret_cast<const nvstrdesc_s *>(src8)->ptr;
            uint32_t v           = len - 4;  // string length
            dst[pos + 0]         = v;
            dst[pos + 1]         = v >> 8;
            dst[pos + 2]         = v >> 16;
            dst[pos + 3]         = v >> 24;
            if (v != 0) memcpy(dst + pos + 4, str_data, v);
          } break;
        }
      }
      __syncthreads();
    }
  }
  if (t == 0) {
    uint8_t *base                = s->page.page_data + s->page.max_hdr_size;
    uint32_t actual_data_size    = static_cast<uint32_t>(s->cur - base);
    uint32_t compressed_bfr_size = GetMaxCompressedBfrSize(actual_data_size);
    s->page.max_data_size        = actual_data_size;
    s->comp_in.srcDevice         = base;
    s->comp_in.srcSize           = actual_data_size;
    s->comp_in.dstDevice         = s->page.compressed_data + s->page.max_hdr_size;
    s->comp_in.dstSize           = compressed_bfr_size;
    s->comp_out.bytes_written    = 0;
    s->comp_out.status           = ~0;
    s->comp_out.reserved         = 0;
  }
  __syncthreads();
  if (t == 0) {
    pages[start_page + blockIdx.x] = s->page;
    if (comp_in) comp_in[blockIdx.x] = s->comp_in;
    if (comp_out) comp_out[blockIdx.x] = s->comp_out;
  }
}

// blockDim(128, 1, 1)
__global__ void __launch_bounds__(128) gpuDecideCompression(EncColumnChunk *chunks,
                                                            const EncPage *pages,
                                                            const gpu_inflate_status_s *comp_out,
                                                            uint32_t start_page)
{
  __shared__ __align__(8) EncColumnChunk ck_g;
  __shared__ __align__(4) unsigned int error_count;
  using warp_reduce = hipcub::WarpReduce<uint32_t>;
  __shared__ typename warp_reduce::TempStorage temp_storage[2];

  uint32_t t                      = threadIdx.x;
  uint32_t uncompressed_data_size = 0;
  uint32_t compressed_data_size   = 0;
  uint32_t first_page, num_pages;

  if (t == 0) {
    ck_g = chunks[blockIdx.x];
    atomicAnd(&error_count, 0);
  }
  __syncthreads();
  if (t < 32) {
    first_page = ck_g.first_page;
    num_pages  = ck_g.num_pages;
    for (uint32_t page = t; page < num_pages; page += 32) {
      uint32_t page_data_size = pages[first_page + page].max_data_size;
      uint32_t comp_idx       = first_page + page - start_page;
      uncompressed_data_size += page_data_size;
      if (comp_out) {
        compressed_data_size += (uint32_t)comp_out[comp_idx].bytes_written;
        if (comp_out[comp_idx].status != 0) { atomicAdd(&error_count, 1); }
      }
    }
    uncompressed_data_size = warp_reduce(temp_storage[0]).Sum(uncompressed_data_size);
    compressed_data_size   = warp_reduce(temp_storage[1]).Sum(compressed_data_size);
  }
  __syncthreads();
  if (t == 0) {
    bool is_compressed;
    if (comp_out) {
      uint32_t compression_error = atomicAdd(&error_count, 0);
      is_compressed = (!compression_error && compressed_data_size < uncompressed_data_size);
    } else {
      is_compressed = false;
    }
    chunks[blockIdx.x].is_compressed = is_compressed;
    chunks[blockIdx.x].bfr_size      = uncompressed_data_size;
    chunks[blockIdx.x].compressed_size =
      (is_compressed) ? compressed_data_size : uncompressed_data_size;
  }
}

/**
 * Minimal thrift compact protocol support
 **/
inline __device__ uint8_t *cpw_put_uint32(uint8_t *p, uint32_t v)
{
  while (v > 0x7f) {
    *p++ = v | 0x80;
    v >>= 7;
  }
  *p++ = v;
  return p;
}

inline __device__ uint8_t *cpw_put_uint64(uint8_t *p, uint64_t v)
{
  while (v > 0x7f) {
    *p++ = v | 0x80;
    v >>= 7;
  }
  *p++ = v;
  return p;
}

inline __device__ uint8_t *cpw_put_int32(uint8_t *p, int32_t v)
{
  int32_t s = (v < 0);
  return cpw_put_uint32(p, (v ^ -s) * 2 + s);
}

inline __device__ uint8_t *cpw_put_int64(uint8_t *p, int64_t v)
{
  int64_t s = (v < 0);
  return cpw_put_uint64(p, (v ^ -s) * 2 + s);
}

inline __device__ uint8_t *cpw_put_fldh(uint8_t *p, int f, int cur, int t)
{
  if (f > cur && f <= cur + 15) {
    *p++ = ((f - cur) << 4) | t;
    return p;
  } else {
    *p++ = t;
    return cpw_put_int32(p, f);
  }
}

class header_encoder {
  uint8_t *current_header_ptr;
  int current_field_index;

 public:
  inline __device__ header_encoder(uint8_t *header_start)
    : current_header_ptr(header_start), current_field_index(0)
  {
  }

  inline __device__ void field_struct_begin(int field)
  {
    current_header_ptr =
      cpw_put_fldh(current_header_ptr, field, current_field_index, ST_FLD_STRUCT);
    current_field_index = 0;
  }

  inline __device__ void field_struct_end(int field)
  {
    *current_header_ptr++ = 0;
    current_field_index   = field;
  }

  template <typename T>
  inline __device__ void field_int32(int field, T value)
  {
    current_header_ptr  = cpw_put_fldh(current_header_ptr, field, current_field_index, ST_FLD_I32);
    current_header_ptr  = cpw_put_int32(current_header_ptr, static_cast<int32_t>(value));
    current_field_index = field;
  }

  template <typename T>
  inline __device__ void field_int64(int field, T value)
  {
    current_header_ptr  = cpw_put_fldh(current_header_ptr, field, current_field_index, ST_FLD_I64);
    current_header_ptr  = cpw_put_int64(current_header_ptr, static_cast<int64_t>(value));
    current_field_index = field;
  }

  inline __device__ void field_binary(int field, const void *value, uint32_t length)
  {
    current_header_ptr =
      cpw_put_fldh(current_header_ptr, field, current_field_index, ST_FLD_BINARY);
    current_header_ptr = cpw_put_uint32(current_header_ptr, length);
    memcpy(current_header_ptr, value, length);
    current_header_ptr += length;
    current_field_index = field;
  }

  inline __device__ void end(uint8_t **header_end, bool termination_flag = true)
  {
    if (termination_flag == false) { *current_header_ptr++ = 0; }
    *header_end = current_header_ptr;
  }

  inline __device__ uint8_t *get_ptr(void) { return current_header_ptr; }

  inline __device__ void set_ptr(uint8_t *ptr) { current_header_ptr = ptr; }
};

__device__ uint8_t *EncodeStatistics(uint8_t *start,
                                     const statistics_chunk *s,
                                     const EncColumnDesc *col,
                                     float *fp_scratch)
{
  uint8_t *end, dtype, dtype_len;
  dtype = col->stats_dtype;
  switch (dtype) {
    case dtype_bool: dtype_len = 1; break;
    case dtype_int8:
    case dtype_int16:
    case dtype_int32:
    case dtype_date32:
    case dtype_float32: dtype_len = 4; break;
    case dtype_int64:
    case dtype_timestamp64:
    case dtype_float64:
    case dtype_decimal64: dtype_len = 8; break;
    case dtype_decimal128: dtype_len = 16; break;
    case dtype_string:
    default: dtype_len = 0; break;
  }
  header_encoder encoder(start);
  encoder.field_int64(3, s->null_count);
  if (s->has_minmax) {
    const void *vmin, *vmax;
    uint32_t lmin, lmax;

    if (dtype == dtype_string) {
      lmin = s->min_value.str_val.length;
      vmin = s->min_value.str_val.ptr;
      lmax = s->max_value.str_val.length;
      vmax = s->max_value.str_val.ptr;
    } else {
      lmin = lmax = dtype_len;
      if (dtype == dtype_float32) {  // Convert from double to float32
        fp_scratch[0] = s->min_value.fp_val;
        fp_scratch[1] = s->max_value.fp_val;
        vmin          = &fp_scratch[0];
        vmax          = &fp_scratch[1];
      } else {
        vmin = &s->min_value;
        vmax = &s->max_value;
      }
    }
    encoder.field_binary(5, vmax, lmax);
    encoder.field_binary(6, vmin, lmin);
  }
  encoder.end(&end);
  return end;
}

// blockDim(128, 1, 1)
__global__ void __launch_bounds__(128) gpuEncodePageHeaders(EncPage *pages,
                                                            EncColumnChunk *chunks,
                                                            const gpu_inflate_status_s *comp_out,
                                                            const statistics_chunk *page_stats,
                                                            const statistics_chunk *chunk_stats,
                                                            uint32_t start_page)
{
  __shared__ __align__(8) EncColumnDesc col_g;
  __shared__ __align__(8) EncColumnChunk ck_g;
  __shared__ __align__(8) EncPage page_g;
  __shared__ __align__(8) float fp_scratch[2];

  uint32_t t = threadIdx.x;

  if (t == 0) {
    uint8_t *hdr_start, *hdr_end;
    uint32_t compressed_page_size, uncompressed_page_size;

    page_g = pages[start_page + blockIdx.x];
    ck_g   = chunks[page_g.chunk_id];
    col_g  = *ck_g.col_desc;

    if (chunk_stats && start_page + blockIdx.x == ck_g.first_page) {
      hdr_start = (ck_g.is_compressed) ? ck_g.compressed_bfr : ck_g.uncompressed_bfr;
      hdr_end   = EncodeStatistics(hdr_start, &chunk_stats[page_g.chunk_id], &col_g, fp_scratch);
      chunks[page_g.chunk_id].ck_stat_size = static_cast<uint32_t>(hdr_end - hdr_start);
    }
    uncompressed_page_size = page_g.max_data_size;
    if (ck_g.is_compressed) {
      hdr_start            = page_g.compressed_data;
      compressed_page_size = (uint32_t)comp_out[blockIdx.x].bytes_written;
      page_g.max_data_size = compressed_page_size;
    } else {
      hdr_start            = page_g.page_data;
      compressed_page_size = uncompressed_page_size;
    }
    header_encoder encoder(hdr_start);
    PageType page_type = page_g.page_type;
    // NOTE: For dictionary encoding, parquet v2 recommends using PLAIN in dictionary page and
    // RLE_DICTIONARY in data page, but parquet v1 uses PLAIN_DICTIONARY in both dictionary and
    // data pages (actual encoding is identical).
    Encoding encoding;
    if (enable_bool_rle) {
      encoding = (col_g.physical_type != BOOLEAN)
                   ? (page_type == PageType::DICTIONARY_PAGE || page_g.dict_bits_plus1 != 0)
                       ? Encoding::PLAIN_DICTIONARY
                       : Encoding::PLAIN
                   : Encoding::RLE;
    } else {
      encoding = (page_type == PageType::DICTIONARY_PAGE || page_g.dict_bits_plus1 != 0)
                   ? Encoding::PLAIN_DICTIONARY
                   : Encoding::PLAIN;
    }
    encoder.field_int32(1, page_type);
    encoder.field_int32(2, uncompressed_page_size);
    encoder.field_int32(3, compressed_page_size);
    if (page_type == PageType::DATA_PAGE) {
      // DataPageHeader
      encoder.field_struct_begin(5);
      encoder.field_int32(1, page_g.num_values);  // NOTE: num_values != num_rows for list types
      encoder.field_int32(2, encoding);           // encoding
      encoder.field_int32(3, Encoding::RLE);      // definition_level_encoding
      encoder.field_int32(4, Encoding::RLE);      // repetition_level_encoding
      // Optionally encode page-level statistics
      if (page_stats) {
        encoder.field_struct_begin(5);
        encoder.set_ptr(EncodeStatistics(
          encoder.get_ptr(), &page_stats[start_page + blockIdx.x], &col_g, fp_scratch));
        encoder.field_struct_end(5);
      }
      encoder.field_struct_end(5);
    } else {
      // DictionaryPageHeader
      encoder.field_struct_begin(7);
      encoder.field_int32(1, ck_g.total_dict_entries);  // number of values in dictionary
      encoder.field_int32(2, encoding);
      encoder.field_struct_end(7);
    }
    encoder.end(&hdr_end, false);
    page_g.hdr_size = (uint32_t)(hdr_end - hdr_start);
  }
  __syncthreads();
  if (t == 0) pages[start_page + blockIdx.x] = page_g;
}

// blockDim(1024, 1, 1)
__global__ void __launch_bounds__(1024) gpuGatherPages(EncColumnChunk *chunks, const EncPage *pages)
{
  __shared__ __align__(8) EncColumnChunk ck_g;
  __shared__ __align__(8) EncPage page_g;

  uint32_t t = threadIdx.x;
  uint8_t *dst, *dst_base;
  const EncPage *first_page;
  uint32_t num_pages, uncompressed_size;

  if (t == 0) ck_g = chunks[blockIdx.x];
  __syncthreads();

  first_page = &pages[ck_g.first_page];
  num_pages  = ck_g.num_pages;
  dst        = (ck_g.is_compressed) ? ck_g.compressed_bfr : ck_g.uncompressed_bfr;
  dst += ck_g.ck_stat_size;  // Skip over chunk statistics
  dst_base          = dst;
  uncompressed_size = ck_g.bfr_size;
  for (uint32_t page = 0; page < num_pages; page++) {
    const uint8_t *src;
    uint32_t hdr_len, data_len;

    if (t == 0) { page_g = first_page[page]; }
    __syncthreads();

    src = (ck_g.is_compressed) ? page_g.compressed_data : page_g.page_data;
    // Copy page header
    hdr_len = page_g.hdr_size;
    memcpy_block<1024, true>(dst, src, hdr_len, t);
    src += page_g.max_hdr_size;
    dst += hdr_len;
    // Copy page data
    uncompressed_size += hdr_len;
    data_len = page_g.max_data_size;
    memcpy_block<1024, true>(dst, src, data_len, t);
    dst += data_len;
    __syncthreads();
    if (!t && page == 0 && ck_g.has_dictionary) { ck_g.dictionary_size = hdr_len + data_len; }
  }
  if (t == 0) {
    chunks[blockIdx.x].bfr_size        = uncompressed_size;
    chunks[blockIdx.x].compressed_size = (dst - dst_base);
    if (ck_g.has_dictionary) { chunks[blockIdx.x].dictionary_size = ck_g.dictionary_size; }
  }
}

/**
 * @brief Get the dremel offsets and repetition and definition levels for a LIST column
 *
 * The repetition and definition level values are ideally computed using a recursive call over a
 * nested structure but in order to better utilize GPU resources, this function calculates them
 * with a bottom up merge method.
 *
 * Given a LIST column of type `List<List<int>>` like so:
 * ```
 * col = {
 *    [],
 *    [[], [1, 2, 3], [4, 5]],
 *    [[]]
 * }
 * ```
 * We can represent it in cudf format with two level of offsets like this:
 * ```
 * Level 0 offsets = {0, 0, 3, 5, 6}
 * Level 1 offsets = {0, 0, 3, 5, 5}
 * Values          = {1, 2, 3, 4, 5}
 * ```
 * The desired result of this function is the repetition and definition level values that
 * correspond to the data values:
 * ```
 * col = {[], [[], [1, 2, 3], [4, 5]], [[]]}
 * def = { 0    1,  2, 2, 2,   2, 2,     1 }
 * rep = { 0,   0,  0, 2, 2,   1, 2,     0 }
 * ```
 *
 * Since repetition and definition levels arrays contain a value for each empty list, the size of
 * the rep/def level array can be given by
 * ```
 * rep_level.size() = size of leaf column + number of empty lists in level 0
 *                                        + number of empty lists in level 1 ...
 * ```
 *
 * We start with finding the empty lists in the penultimate level and merging it with the indices
 * of the leaf level. The values for the merge are the definition and repetition levels
 * ```
 * empties at level 1 = {0, 5}
 * def values at 1    = {1, 1}
 * rep values at 1    = {1, 1}
 * indices at leaf    = {0, 1, 2, 3, 4}
 * def values at leaf = {2, 2, 2, 2, 2}
 * rep values at leaf = {2, 2, 2, 2, 2}
 * ```
 *
 * merged def values  = {1, 2, 2, 2, 2, 2, 1}
 * merged rep values  = {1, 2, 2, 2, 2, 2, 1}
 *
 * The size of the rep/def values is now larger than the leaf values and the offsets need to be
 * adjusted in order to point to the correct start indices. We do this with an exclusive scan over
 * the indices of offsets of empty lists and adding to existing offsets.
 * ```
 * Level 1 new offsets = {0, 1, 4, 6, 7}
 * ```
 * Repetition values at the beginning of a list need to be decremented. We use the new offsets to
 * scatter the rep value.
 * ```
 * merged rep values  = {1, 2, 2, 2, 2, 2, 1}
 * scatter (1, new offsets)
 * new offsets        = {0, 1,       4,    6, 7}
 * new rep values     = {1, 1, 2, 2, 1, 2, 1}
 * ```
 *
 * Similarly we merge up all the way till level 0 offsets
 */
dremel_data get_dremel_data(column_view h_col, hipStream_t stream)
{
  CUDF_EXPECTS(h_col.type().id() == type_id::LIST,
               "Can only get rep/def levels for LIST type column");

  auto get_empties = [&](column_view col, size_type start, size_type end) {
    auto lcv = lists_column_view(col);
    rmm::device_uvector<size_type> empties_idx(lcv.size(), stream);
    rmm::device_uvector<size_type> empties(lcv.size(), stream);
    auto d_off = lcv.offsets().data<size_type>();

    auto empties_idx_end =
      thrust::copy_if(rmm::exec_policy(stream)->on(stream),
                      thrust::make_counting_iterator(start),
                      thrust::make_counting_iterator(end),
                      empties_idx.begin(),
                      [d_off] __device__(auto i) { return d_off[i] == d_off[i + 1]; });
    auto empties_end = thrust::gather(rmm::exec_policy(stream)->on(stream),
                                      empties_idx.begin(),
                                      empties_idx_end,
                                      lcv.offsets().begin<size_type>(),
                                      empties.begin());

    auto empties_size = empties_end - empties.begin();
    return std::make_tuple(std::move(empties), std::move(empties_idx), empties_size);
  };

  // Reverse the nesting in order to merge the deepest level with the leaf first and merge bottom
  // up
  auto curr_col        = h_col;
  size_t max_vals_size = 0;
  std::vector<column_view> nesting_levels;
  std::vector<uint8_t> def_at_level;
  while (curr_col.type().id() == type_id::LIST) {
    nesting_levels.push_back(curr_col);
    def_at_level.push_back(curr_col.nullable() ? 2 : 1);
    auto lcv = lists_column_view(curr_col);
    max_vals_size += lcv.offsets().size();
    curr_col = lcv.child();
  }
  // One more entry for leaf col
  def_at_level.push_back(curr_col.nullable() ? 2 : 1);
  max_vals_size += curr_col.size();

  thrust::exclusive_scan(
    thrust::host, def_at_level.begin(), def_at_level.end(), def_at_level.begin());

  // Sliced list column views only have offsets applied to top level. Get offsets for each level.
  hostdevice_vector<size_type> column_offsets(nesting_levels.size() + 1, stream);
  hostdevice_vector<size_type> column_ends(nesting_levels.size() + 1, stream);

  auto d_col = column_device_view::create(h_col, stream);
  cudf::detail::device_single_thread(
    [offset_at_level  = column_offsets.device_ptr(),
     end_idx_at_level = column_ends.device_ptr(),
     col              = *d_col] __device__() {
      auto curr_col           = col;
      size_type off           = curr_col.offset();
      size_type end           = off + curr_col.size();
      size_type level         = 0;
      offset_at_level[level]  = off;
      end_idx_at_level[level] = end;
      ++level;
      // Apply offset recursively until we get to leaf data
      while (curr_col.type().id() == type_id::LIST) {
        off = curr_col.child(lists_column_view::offsets_column_index).element<size_type>(off);
        end = curr_col.child(lists_column_view::offsets_column_index).element<size_type>(end);
        offset_at_level[level]  = off;
        end_idx_at_level[level] = end;
        ++level;
        curr_col = curr_col.child(lists_column_view::child_column_index);
      }
    },
    stream);

  column_offsets.device_to_host(stream, true);
  column_ends.device_to_host(stream, true);

  rmm::device_uvector<uint8_t> rep_level(max_vals_size, stream);
  rmm::device_uvector<uint8_t> def_level(max_vals_size, stream);

  rmm::device_uvector<uint8_t> temp_rep_vals(max_vals_size, stream);
  rmm::device_uvector<uint8_t> temp_def_vals(max_vals_size, stream);
  rmm::device_uvector<size_type> new_offsets(0, stream);
  size_type curr_rep_values_size = 0;
  {
    // At this point, curr_col contains the leaf column. Max nesting level is
    // nesting_levels.size().
    size_t level              = nesting_levels.size() - 1;
    curr_col                  = nesting_levels[level];
    auto lcv                  = lists_column_view(curr_col);
    auto offset_size_at_level = column_ends[level] - column_offsets[level] + 1;

    // Get empties at this level
    rmm::device_uvector<size_type> empties(0, stream);
    rmm::device_uvector<size_type> empties_idx(0, stream);
    size_t empties_size;
    std::tie(empties, empties_idx, empties_size) =
      get_empties(nesting_levels[level], column_offsets[level], column_ends[level]);

    // Merge empty at deepest parent level with the rep, def level vals at leaf level

    auto input_parent_rep_it = thrust::make_constant_iterator(level);
    auto input_parent_def_it = thrust::make_transform_iterator(
      thrust::make_counting_iterator(0),
      [idx            = empties_idx.data(),
       mask           = lcv.null_mask(),
       curr_def_level = def_at_level[level]] __device__(auto i) {
        return curr_def_level + ((mask && bit_is_set(mask, idx[i])) ? 1 : 0);
      });

    auto input_child_rep_it = thrust::make_constant_iterator(nesting_levels.size());
    auto input_child_def_it = thrust::make_transform_iterator(
      thrust::make_counting_iterator(column_offsets[level + 1]),
      [mask = lcv.child().null_mask(), curr_def_level = def_at_level[level + 1]] __device__(
        auto i) { return curr_def_level + ((mask && bit_is_set(mask, i)) ? 1 : 0); });

    // Zip the input and output value iterators so that merge operation is done only once
    auto input_parent_zip_it =
      thrust::make_zip_iterator(thrust::make_tuple(input_parent_rep_it, input_parent_def_it));

    auto input_child_zip_it =
      thrust::make_zip_iterator(thrust::make_tuple(input_child_rep_it, input_child_def_it));

    auto output_zip_it =
      thrust::make_zip_iterator(thrust::make_tuple(rep_level.begin(), def_level.begin()));

    auto ends = thrust::merge_by_key(rmm::exec_policy(stream)->on(stream),
                                     empties.begin(),
                                     empties.begin() + empties_size,
                                     thrust::make_counting_iterator(column_offsets[level + 1]),
                                     thrust::make_counting_iterator(column_ends[level + 1]),
                                     input_parent_zip_it,
                                     input_child_zip_it,
                                     thrust::make_discard_iterator(),
                                     output_zip_it);

    curr_rep_values_size = ends.second - output_zip_it;

    // Scan to get distance by which each offset value is shifted due to the insertion of empties
    auto scan_it =
      thrust::make_transform_iterator(thrust::make_counting_iterator(column_offsets[level]),
                                      [off = lcv.offsets().data<size_type>()] __device__(
                                        auto i) -> int { return off[i] == off[i + 1]; });
    rmm::device_uvector<size_type> scan_out(offset_size_at_level, stream);
    thrust::exclusive_scan(rmm::exec_policy(stream)->on(stream),
                           scan_it,
                           scan_it + offset_size_at_level,
                           scan_out.begin());

    // Add scan output to existing offsets to get new offsets into merged rep level values
    new_offsets = rmm::device_uvector<size_type>(offset_size_at_level, stream);
    thrust::for_each_n(rmm::exec_policy(stream)->on(stream),
                       thrust::make_counting_iterator(0),
                       offset_size_at_level,
                       [off      = lcv.offsets().data<size_type>() + column_offsets[level],
                        scan_out = scan_out.data(),
                        new_off  = new_offsets.data()] __device__(auto i) {
                         new_off[i] = off[i] - off[0] + scan_out[i];
                       });

    // Set rep level values at level starts to appropriate rep level
    auto scatter_it = thrust::make_constant_iterator(level);
    thrust::scatter(rmm::exec_policy(stream)->on(stream),
                    scatter_it,
                    scatter_it + new_offsets.size() - 1,
                    new_offsets.begin(),
                    rep_level.begin());
  }

  for (int level = nesting_levels.size() - 2; level >= 0; level--) {
    curr_col                  = nesting_levels[level];
    auto lcv                  = lists_column_view(curr_col);
    auto offset_size_at_level = column_ends[level] - column_offsets[level] + 1;

    // Get empties at this level
    rmm::device_uvector<size_type> empties(0, stream);
    rmm::device_uvector<size_type> empties_idx(0, stream);
    size_t empties_size;
    std::tie(empties, empties_idx, empties_size) =
      get_empties(nesting_levels[level], column_offsets[level], column_ends[level]);

    auto offset_transformer = [new_child_offsets = new_offsets.data(),
                               child_start       = column_offsets[level + 1]] __device__(auto x) {
      return new_child_offsets[x - child_start];  // (x - child's offset)
    };

    // We will be reading from old rep_levels and writing again to rep_levels. Swap the current
    // rep values into temp_rep_vals so it can become the input and rep_levels can again be output.
    std::swap(temp_rep_vals, rep_level);
    std::swap(temp_def_vals, def_level);

    // Merge empty at parent level with the rep, def level vals at current level
    auto transformed_empties = thrust::make_transform_iterator(empties.begin(), offset_transformer);

    auto input_parent_rep_it = thrust::make_constant_iterator(level);
    auto input_parent_def_it = thrust::make_transform_iterator(
      thrust::make_counting_iterator(0),
      [idx            = empties_idx.data(),
       mask           = lcv.null_mask(),
       curr_def_level = def_at_level[level]] __device__(auto i) {
        return curr_def_level + ((mask && bit_is_set(mask, idx[i])) ? 1 : 0);
      });

    // Zip the input and output value iterators so that merge operation is done only once
    auto input_parent_zip_it =
      thrust::make_zip_iterator(thrust::make_tuple(input_parent_rep_it, input_parent_def_it));

    auto input_child_zip_it =
      thrust::make_zip_iterator(thrust::make_tuple(temp_rep_vals.begin(), temp_def_vals.begin()));

    auto output_zip_it =
      thrust::make_zip_iterator(thrust::make_tuple(rep_level.begin(), def_level.begin()));

    auto ends = thrust::merge_by_key(rmm::exec_policy(stream)->on(stream),
                                     transformed_empties,
                                     transformed_empties + empties_size,
                                     thrust::make_counting_iterator(0),
                                     thrust::make_counting_iterator(curr_rep_values_size),
                                     input_parent_zip_it,
                                     input_child_zip_it,
                                     thrust::make_discard_iterator(),
                                     output_zip_it);

    curr_rep_values_size = ends.second - output_zip_it;

    // Scan to get distance by which each offset value is shifted due to the insertion of dremel
    // level value fof an empty list
    auto scan_it =
      thrust::make_transform_iterator(thrust::make_counting_iterator(column_offsets[level]),
                                      [off = lcv.offsets().data<size_type>()] __device__(
                                        auto i) -> int { return off[i] == off[i + 1]; });
    rmm::device_uvector<size_type> scan_out(offset_size_at_level, stream);
    thrust::exclusive_scan(rmm::exec_policy(stream)->on(stream),
                           scan_it,
                           scan_it + offset_size_at_level,
                           scan_out.begin());

    // Add scan output to existing offsets to get new offsets into merged rep level values
    rmm::device_uvector<size_type> temp_new_offsets(offset_size_at_level, stream);
    thrust::for_each_n(rmm::exec_policy(stream)->on(stream),
                       thrust::make_counting_iterator(0),
                       offset_size_at_level,
                       [off      = lcv.offsets().data<size_type>() + column_offsets[level],
                        scan_out = scan_out.data(),
                        new_off  = temp_new_offsets.data(),
                        offset_transformer] __device__(auto i) {
                         new_off[i] = offset_transformer(off[i]) + scan_out[i];
                       });
    new_offsets = std::move(temp_new_offsets);

    // Set rep level values at level starts to appropriate rep level
    auto scatter_it = thrust::make_constant_iterator(level);
    thrust::scatter(rmm::exec_policy(stream)->on(stream),
                    scatter_it,
                    scatter_it + new_offsets.size() - 1,
                    new_offsets.begin(),
                    rep_level.begin());
  }

  size_t level_vals_size = new_offsets.back_element(stream);
  rep_level.resize(level_vals_size, stream);
  def_level.resize(level_vals_size, stream);

  CUDA_TRY(hipStreamSynchronize(stream));

  size_type leaf_col_offset = column_offsets[column_offsets.size() - 1];
  size_type leaf_data_size  = column_ends[column_ends.size() - 1] - leaf_col_offset;

  return dremel_data{std::move(new_offsets),
                     std::move(rep_level),
                     std::move(def_level),
                     leaf_col_offset,
                     leaf_data_size};
}

/**
 * @brief Launches kernel for initializing encoder page fragments
 *
 * @param[in,out] frag Fragment array [column_id][fragment_id]
 * @param[in] col_desc Column description array [column_id]
 * @param[in] num_fragments Number of fragments per column
 * @param[in] num_columns Number of columns
 * @param[in] stream CUDA stream to use, default 0
 */
void InitPageFragments(PageFragment *frag,
                       const EncColumnDesc *col_desc,
                       int32_t num_fragments,
                       int32_t num_columns,
                       uint32_t fragment_size,
                       uint32_t num_rows,
                       hipStream_t stream)
{
  dim3 dim_grid(num_columns, num_fragments);  // 1 threadblock per fragment
  gpuInitPageFragments<512><<<dim_grid, 512, 0, stream>>>(
    frag, col_desc, num_fragments, num_columns, fragment_size, num_rows);
}

/**
 * @brief Launches kernel for initializing fragment statistics groups
 *
 * @param[out] groups Statistics groups [num_columns x num_fragments]
 * @param[in] fragments Page fragments [num_columns x num_fragments]
 * @param[in] col_desc Column description [num_columns]
 * @param[in] num_fragments Number of fragments
 * @param[in] num_columns Number of columns
 * @param[in] fragment_size Max size of each fragment in rows
 * @param[in] stream CUDA stream to use, default 0
 */
void InitFragmentStatistics(statistics_group *groups,
                            const PageFragment *fragments,
                            const EncColumnDesc *col_desc,
                            int32_t num_fragments,
                            int32_t num_columns,
                            uint32_t fragment_size,
                            hipStream_t stream)
{
  dim3 dim_grid(num_columns, (num_fragments + 3) >> 2);  // 1 warp per fragment
  gpuInitFragmentStats<<<dim_grid, 128, 0, stream>>>(
    groups, fragments, col_desc, num_fragments, num_columns, fragment_size);
}

/**
 * @brief Launches kernel for initializing encoder data pages
 *
 * @param[in,out] chunks Column chunks [rowgroup][column]
 * @param[out] pages Encode page array (null if just counting pages)
 * @param[in] col_desc Column description array [column_id]
 * @param[in] num_rowgroups Number of fragments per column
 * @param[in] num_columns Number of columns
 * @param[out] page_grstats Setup for page-level stats
 * @param[out] chunk_grstats Setup for chunk-level stats
 * @param[in] stream CUDA stream to use, default 0
 */
void InitEncoderPages(EncColumnChunk *chunks,
                      EncPage *pages,
                      const EncColumnDesc *col_desc,
                      int32_t num_rowgroups,
                      int32_t num_columns,
                      statistics_merge_group *page_grstats,
                      statistics_merge_group *chunk_grstats,
                      hipStream_t stream)
{
  dim3 dim_grid(num_columns, num_rowgroups);  // 1 threadblock per rowgroup
  gpuInitPages<<<dim_grid, 128, 0, stream>>>(
    chunks, pages, col_desc, page_grstats, chunk_grstats, num_rowgroups, num_columns);
}

/**
 * @brief Launches kernel for packing column data into parquet pages
 *
 * @param[in,out] pages Device array of EncPages (unordered)
 * @param[in] chunks Column chunks
 * @param[in] num_pages Number of pages
 * @param[in] start_page First page to encode in page array
 * @param[out] comp_in Optionally initializes compressor input params
 * @param[out] comp_out Optionally initializes compressor output params
 * @param[in] stream CUDA stream to use, default 0
 */
void EncodePages(EncPage *pages,
                 const EncColumnChunk *chunks,
                 uint32_t num_pages,
                 uint32_t start_page,
                 gpu_inflate_input_s *comp_in,
                 gpu_inflate_status_s *comp_out,
                 hipStream_t stream)
{
  // A page is part of one column. This is launching 1 block per page. 1 block will exclusively
  // deal with one datatype.
  gpuEncodePages<<<num_pages, 128, 0, stream>>>(pages, chunks, comp_in, comp_out, start_page);
}

/**
 * @brief Launches kernel to make the compressed vs uncompressed chunk-level decision
 *
 * @param[in,out] chunks Column chunks
 * @param[in] pages Device array of EncPages (unordered)
 * @param[in] num_chunks Number of column chunks
 * @param[in] start_page First page to encode in page array
 * @param[in] comp_out Compressor status
 * @param[in] stream CUDA stream to use, default 0
 */
void DecideCompression(EncColumnChunk *chunks,
                       const EncPage *pages,
                       uint32_t num_chunks,
                       uint32_t start_page,
                       const gpu_inflate_status_s *comp_out,
                       hipStream_t stream)
{
  gpuDecideCompression<<<num_chunks, 128, 0, stream>>>(chunks, pages, comp_out, start_page);
}

/**
 * @brief Launches kernel to encode page headers
 *
 * @param[in,out] pages Device array of EncPages
 * @param[in,out] chunks Column chunks
 * @param[in] num_pages Number of pages
 * @param[in] start_page First page to encode in page array
 * @param[in] comp_out Compressor status or nullptr if no compression
 * @param[in] page_stats Optional page-level statistics to be included in page header
 * @param[in] chunk_stats Optional chunk-level statistics to be encoded
 * @param[in] stream CUDA stream to use, default 0
 */
void EncodePageHeaders(EncPage *pages,
                       EncColumnChunk *chunks,
                       uint32_t num_pages,
                       uint32_t start_page,
                       const gpu_inflate_status_s *comp_out,
                       const statistics_chunk *page_stats,
                       const statistics_chunk *chunk_stats,
                       hipStream_t stream)
{
  gpuEncodePageHeaders<<<num_pages, 128, 0, stream>>>(
    pages, chunks, comp_out, page_stats, chunk_stats, start_page);
}

/**
 * @brief Launches kernel to gather pages to a single contiguous block per chunk
 *
 * @param[in,out] chunks Column chunks
 * @param[in] pages Device array of EncPages
 * @param[in] num_chunks Number of column chunks
 * @param[in] stream CUDA stream to use, default 0
 */
void GatherPages(EncColumnChunk *chunks,
                 const EncPage *pages,
                 uint32_t num_chunks,
                 hipStream_t stream)
{
  gpuGatherPages<<<num_chunks, 1024, 0, stream>>>(chunks, pages);
}

}  // namespace gpu
}  // namespace parquet
}  // namespace io
}  // namespace cudf
