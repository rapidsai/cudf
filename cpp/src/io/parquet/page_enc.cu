#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */
#include "parquet_gpu.h"
#include <io/utilities/block_utils.cuh>

namespace cudf {
namespace io {
namespace parquet {
namespace gpu {

struct frag_init_state_s
{
    EncColumnDesc col;
    PageFragment frag;
    volatile uint32_t scratch_red[32];
};


/**
 * @brief Initializes encoder page fragments
 *
 * @param[in] frag Fragment array [fragment_id][column_id]
 * @param[in] col_desc Column description array [column_id]
 * @param[in] num_fragments Number of fragments per column
 * @param[in] num_columns Number of columns
 *
 **/
// blockDim {512,1,1}
__global__ void __launch_bounds__(512)
gpuInitPageFragments(PageFragment *frag, const EncColumnDesc *col_desc, int32_t num_fragments, int32_t num_columns, uint32_t fragment_size, uint32_t max_num_rows)
{
    __shared__ __align__(16) frag_init_state_s state_g;

    frag_init_state_s * const s = &state_g;
    uint32_t t = threadIdx.x;
    uint32_t start_row, nrows, dtype_len, dtype;

    if (t < sizeof(EncColumnDesc) / sizeof(uint32_t))
    {
        reinterpret_cast<uint32_t *>(&s->col)[t] = reinterpret_cast<const uint32_t *>(&col_desc[blockIdx.x])[t];
    }
    __syncthreads();
    start_row = blockIdx.y * fragment_size;
    if (!t)
    {
        s->col.num_rows = min(s->col.num_rows, max_num_rows);
        s->frag.num_rows = min(fragment_size, max_num_rows - min(start_row, max_num_rows));
        s->frag.non_nulls = 0;
        s->frag.fragment_data_size = 0;
    }
    dtype = s->col.physical_type;
    dtype_len = (dtype == INT64 || dtype == INT64) ? 8 : (dtype == BOOLEAN) ? 1 : 4;
    __syncthreads();
    nrows = s->frag.num_rows;
    for (uint32_t i = 0; i < nrows; i += 512)
    {
        const uint32_t *valid = s->col.valid_map_base;
        uint32_t row = start_row + i + t;
        uint32_t is_valid = (row < s->col.num_rows) ? (valid[row >> 5] >> (row & 0x1f)) & 1 : (valid) ? 1 : 0;
        uint32_t valid_warp = BALLOT(is_valid);
        uint32_t len;
        if (is_valid) {
            len = dtype_len;
            if (dtype == BYTE_ARRAY) {
                len += (uint32_t)reinterpret_cast<const nvstrdesc_s *>(s->col.column_data_base)[row].count;
            }
        } else {
            len = 0;
        }
        len = WarpReduceSum32(len);
        if (!(t & 0x1f)) {
            s->scratch_red[(t >> 5) + 0] = __popc(valid_warp);
            s->scratch_red[(t >> 5) + 16] = len;
        }
        __syncthreads();
        if (t < 32) {
            uint32_t non_nulls = WarpReduceSum16((t < 16) ? s->scratch_red[t] : 0);
            len = WarpReduceSum16((t < 16) ? s->scratch_red[t + 16] : 0);
            if (!t) {
                s->frag.non_nulls = s->frag.non_nulls + non_nulls;
                s->frag.fragment_data_size += len;
            }
        }
    }
    __syncthreads();
    if (t < sizeof(PageFragment) / sizeof(uint32_t)) {
        reinterpret_cast<uint32_t *>(&frag[blockIdx.x * num_fragments + blockIdx.y])[t] = reinterpret_cast<uint32_t *>(&s->frag)[t];
    }
}

// blockDim {128,1,1}
__global__ void __launch_bounds__(128)
gpuInitPages(EncColumnChunk *chunks, EncPage *pages, const EncColumnDesc *col_desc, int32_t num_rowgroups, int32_t num_columns)
{
    //__shared__ __align__(8) EncColumnDesc col_g;
    __shared__ __align__(8) EncColumnChunk ck_g;
    __shared__ __align__(8) PageFragment frag_g;
    __shared__ __align__(8) EncPage page_g;

    uint32_t t = threadIdx.x;
    
    /*if (t < sizeof(EncColumnDesc) / sizeof(uint32_t)) {
        reinterpret_cast<uint32_t *>(&col_g)[t] = reinterpret_cast<const uint32_t *>(&col_desc[blockIdx.x])[t];
    }*/
    if (t < sizeof(EncColumnChunk) / sizeof(uint32_t)) {
        reinterpret_cast<uint32_t *>(&ck_g)[t] = reinterpret_cast<const uint32_t *>(&chunks[blockIdx.y * num_columns + blockIdx.x])[t];
    }
    __syncthreads();
    if (t < 32) {
        uint32_t fragments_in_chunk = 0;
        uint32_t rows_in_page = 0;
        uint32_t page_size = 0;
        uint32_t num_pages = 0;
        uint32_t num_rows = 0;
        uint32_t page_start = 0;
        do {
            uint32_t fragment_data_size, max_page_size;
            SYNCWARP();
            if (num_rows < ck_g.num_rows) {
                if (t < sizeof(PageFragment) / sizeof(uint32_t)) {
                    reinterpret_cast<uint32_t *>(&frag_g)[t] = reinterpret_cast<const uint32_t *>(&ck_g.fragments[fragments_in_chunk])[t];
                }
            } else if (!t) {
                frag_g.fragment_data_size = 0;
                frag_g.num_rows = 0;
            }
            SYNCWARP();
            fragment_data_size = frag_g.fragment_data_size;
            max_page_size = (rows_in_page * 2 >= ck_g.num_rows) ? 256 * 1024 : (rows_in_page * 3 >= ck_g.num_rows) ? 384 * 1024 : 512 * 1024;
            if (num_rows >= ck_g.num_rows || page_size + fragment_data_size > max_page_size)
            {
                if (!t) {
                    page_g.num_fragments = fragments_in_chunk - page_start;
                }
                SYNCWARP();
                if (pages && t < sizeof(EncPage) / sizeof(uint32_t)) {
                    reinterpret_cast<uint32_t *>(&pages[ck_g.first_page + num_pages])[t] = reinterpret_cast<uint32_t *>(&page_g)[t];
                }
                num_pages++;
                page_size = 0;
                rows_in_page = 0;
                page_start = fragments_in_chunk;
            }
            page_size += fragment_data_size;
            rows_in_page += frag_g.num_rows;
            num_rows += frag_g.num_rows;
            fragments_in_chunk++;
        } while (frag_g.num_rows != 0);
        if (!t) {
            ck_g.num_pages = num_pages;
        }
    }
    __syncthreads();
    if (t < sizeof(EncColumnChunk) / sizeof(uint32_t)) {
        reinterpret_cast<uint32_t *>(&chunks[blockIdx.y * num_columns + blockIdx.x])[t] = reinterpret_cast<uint32_t *>(&ck_g)[t];
    }
}


/**
 * @brief Launches kernel for initializing encoder page fragments
 *
 * @param[in] frag Fragment array [column_id][fragment_id]
 * @param[in] col_desc Column description array [column_id]
 * @param[in] num_fragments Number of fragments per column
 * @param[in] num_columns Number of columns
 * @param[in] stream CUDA stream to use, default 0
 *
 * @return hipSuccess if successful, a CUDA error code otherwise
 **/
hipError_t InitPageFragments(PageFragment *frag, const EncColumnDesc *col_desc, int32_t num_fragments, int32_t num_columns, uint32_t fragment_size, uint32_t num_rows, hipStream_t stream)
{
    dim3 dim_grid(num_columns, num_fragments);  // 1 threadblock per fragment
    gpuInitPageFragments <<< dim_grid, 512, 0, stream >>> (frag, col_desc, num_fragments, num_columns, fragment_size, num_rows);
    return hipSuccess;
}


/**
 * @brief Launches kernel for initializing encoder data pages
 *
 * @param[in,out] chunks Column chunks [rowgroup][column]
 * @param[out] pages Encode page array (null if just counting pages)
 * @param[in] col_desc Column description array [column_id]
 * @param[in] num_rowgroups Number of fragments per column
 * @param[in] num_columns Number of columns
 * @param[in] stream CUDA stream to use, default 0
 *
 * @return hipSuccess if successful, a CUDA error code otherwise
 **/
hipError_t InitEncoderPages(EncColumnChunk *chunks, EncPage *pages, const EncColumnDesc *col_desc, int32_t num_rowgroups, int32_t num_columns, hipStream_t stream)
{
    dim3 dim_grid(num_columns, num_rowgroups);  // 1 threadblock per rowgroup
    gpuInitPages <<< dim_grid, 128, 0, stream >>> (chunks, pages, col_desc, num_rowgroups, num_columns);
    return hipSuccess;
}


} // namespace gpu
} // namespace parquet
} // namespace io
} // namespace cudf
