#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019-2024, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "delta_enc.cuh"
#include "io/utilities/block_utils.cuh"
#include "page_string_utils.cuh"
#include "parquet_gpu.cuh"

#include <cudf/detail/iterator.cuh>
#include <cudf/detail/utilities/assert.cuh>
#include <cudf/detail/utilities/cuda.cuh>
#include <cudf/detail/utilities/stream_pool.hpp>
#include <cudf/detail/utilities/vector_factories.hpp>

#include <rmm/cuda_stream_view.hpp>
#include <rmm/exec_policy.hpp>

#include <hipcub/hipcub.hpp>
#include <cuda/std/chrono>
#include <thrust/binary_search.h>
#include <thrust/distance.h>
#include <thrust/gather.h>
#include <thrust/iterator/discard_iterator.h>
#include <thrust/iterator/reverse_iterator.h>
#include <thrust/iterator/transform_iterator.h>
#include <thrust/iterator/zip_iterator.h>
#include <thrust/merge.h>
#include <thrust/scan.h>
#include <thrust/scatter.h>
#include <thrust/tuple.h>

#include <bitset>

namespace cudf::io::parquet::detail {

namespace {

using ::cudf::detail::device_2dspan;

constexpr int encode_block_size = 128;
constexpr int rle_buffer_size   = 2 * encode_block_size;
constexpr int num_encode_warps  = encode_block_size / cudf::detail::warp_size;

constexpr int rolling_idx(int pos) { return rolling_index<rle_buffer_size>(pos); }

// max V1 header size
// also valid for dict page header (V1 or V2)
constexpr int MAX_V1_HDR_SIZE = util::round_up_unsafe(27, 8);

// max V2 header size
constexpr int MAX_V2_HDR_SIZE = util::round_up_unsafe(49, 8);

// do not truncate statistics
constexpr int32_t NO_TRUNC_STATS = 0;

// minimum scratch space required for encoding statistics
constexpr size_t MIN_STATS_SCRATCH_SIZE = sizeof(__int128_t);

// mask to determine lane id
constexpr uint32_t WARP_MASK = cudf::detail::warp_size - 1;

// currently 64k - 1
constexpr uint32_t MAX_GRID_Y_SIZE = (1 << 16) - 1;

// space needed for RLE length field
constexpr int RLE_LENGTH_FIELD_LEN = 4;

struct frag_init_state_s {
  parquet_column_device_view col;
  PageFragment frag;
};

template <int rle_buf_size>
struct page_enc_state_s {
  uint8_t* cur;          //!< current output ptr
  uint8_t* rle_out;      //!< current RLE write ptr
  uint8_t* rle_len_pos;  //!< position to write RLE length (for V2 boolean data)
  uint32_t rle_run;      //!< current RLE run
  uint32_t run_val;      //!< current RLE run value
  uint32_t rle_pos;      //!< RLE encoder positions
  uint32_t rle_numvals;  //!< RLE input value count
  uint32_t rle_lit_count;
  uint32_t rle_rpt_count;
  uint32_t page_start_val;
  uint32_t chunk_start_val;
  uint32_t rpt_map[num_encode_warps];
  EncPage page;
  EncColumnChunk ck;
  parquet_column_device_view col;
  uint32_t vals[rle_buf_size];
};

using rle_page_enc_state_s = page_enc_state_s<rle_buffer_size>;

/**
 * @brief Returns the size of the type in the Parquet file.
 */
constexpr uint32_t physical_type_len(Type physical_type, type_id id)
{
  if (physical_type == FIXED_LEN_BYTE_ARRAY and id == type_id::DECIMAL128) {
    return sizeof(__int128_t);
  }
  switch (physical_type) {
    case INT96: return 12u;
    case INT64:
    case DOUBLE: return sizeof(int64_t);
    case BOOLEAN: return 1u;
    default: return sizeof(int32_t);
  }
}

constexpr uint32_t max_RLE_page_size(uint8_t value_bit_width, uint32_t num_values)
{
  if (value_bit_width == 0) return 0;

  // Run length = 4, max(rle/bitpack header) = 5. bitpacking worst case is one byte every 8 values
  // (because bitpacked runs are a multiple of 8). Don't need to round up the last term since that
  // overhead is accounted for in the '5'.
  // TODO: this formula does not take into account the data for RLE runs. The worst realistic case
  // is repeated runs of 8 bitpacked, 2 RLE values. In this case, the formula would be
  //   0.8 * (num_values * bw / 8 + num_values / 8) + 0.2 * (num_values / 2 * (1 + (bw+7)/8))
  // for bw < 8 the above value will be larger than below, but in testing it seems like for low
  // bitwidths it's hard to get the pathological 8:2 split.
  // If the encoder starts printing the data corruption warning, then this will need to be
  // revisited.
  return 4 + 5 + util::div_rounding_up_unsafe(num_values * value_bit_width, 8) + (num_values / 8);
}

// subtract b from a, but return 0 if this would underflow
constexpr size_t underflow_safe_subtract(size_t a, size_t b)
{
  if (b > a) { return 0; }
  return a - b;
}

void __device__ init_frag_state(frag_init_state_s* const s,
                                uint32_t fragment_size,
                                int part_end_row)
{
  // frag.num_rows = fragment_size except for the last fragment in partition which can be
  // smaller. num_rows is fixed but fragment size could be larger if the data is strings or
  // nested.
  s->frag.num_rows           = min(fragment_size, part_end_row - s->frag.start_row);
  s->frag.num_dict_vals      = 0;
  s->frag.fragment_data_size = 0;
  s->frag.dict_data_size     = 0;

  s->frag.start_value_idx  = row_to_value_idx(s->frag.start_row, s->col);
  auto const end_value_idx = row_to_value_idx(s->frag.start_row + s->frag.num_rows, s->col);
  s->frag.num_leaf_values  = end_value_idx - s->frag.start_value_idx;

  if (s->col.level_offsets != nullptr) {
    // For nested schemas, the number of values in a fragment is not directly related to the
    // number of encoded data elements or the number of rows.  It is simply the number of
    // repetition/definition values which together encode validity and nesting information.
    auto const first_level_val_idx = s->col.level_offsets[s->frag.start_row];
    auto const last_level_val_idx  = s->col.level_offsets[s->frag.start_row + s->frag.num_rows];
    s->frag.num_values             = last_level_val_idx - first_level_val_idx;
  } else {
    s->frag.num_values = s->frag.num_rows;
  }
}

template <int block_size>
void __device__ calculate_frag_size(frag_init_state_s* const s, int t)
{
  using block_reduce = hipcub::BlockReduce<uint32_t, block_size>;
  __shared__ typename block_reduce::TempStorage reduce_storage;

  auto const physical_type   = s->col.physical_type;
  auto const leaf_type       = s->col.leaf_column->type().id();
  auto const dtype_len       = physical_type_len(physical_type, leaf_type);
  auto const nvals           = s->frag.num_leaf_values;
  auto const start_value_idx = s->frag.start_value_idx;

  uint32_t num_valid = 0;
  uint32_t len       = 0;
  for (uint32_t i = 0; i < nvals; i += block_size) {
    auto const val_idx  = start_value_idx + i + t;
    auto const is_valid = i + t < nvals && val_idx < s->col.leaf_column->size() &&
                          s->col.leaf_column->is_valid(val_idx);
    if (is_valid) {
      num_valid++;
      len += dtype_len;
      if (physical_type == BYTE_ARRAY) {
        switch (leaf_type) {
          case type_id::STRING: {
            auto str = s->col.leaf_column->element<string_view>(val_idx);
            len += str.size_bytes();
          } break;
          case type_id::LIST: {
            auto list_element =
              get_element<statistics::byte_array_view>(*s->col.leaf_column, val_idx);
            len += list_element.size_bytes();
          } break;
          default: CUDF_UNREACHABLE("Unsupported data type for leaf column");
        }
      }
    }
  }

  auto const total_len = block_reduce(reduce_storage).Sum(len);
  __syncthreads();
  auto const total_valid = block_reduce(reduce_storage).Sum(num_valid);

  if (t == 0) {
    s->frag.fragment_data_size = total_len;
    s->frag.num_valid          = total_valid;
  }

  __syncthreads();
  // page fragment size must fit in a 32-bit signed integer
  if (s->frag.fragment_data_size > static_cast<uint32_t>(std::numeric_limits<int32_t>::max())) {
    // TODO need to propagate this error back to the host
    CUDF_UNREACHABLE("page fragment size exceeds maximum for i32");
  }
}

/**
 * @brief Determine the correct page encoding for the given page parameters.
 *
 * This is only used by the plain and dictionary encoders. Delta encoders will set the page
 * encoding directly.
 */
Encoding __device__ determine_encoding(PageType page_type,
                                       Type physical_type,
                                       bool use_dictionary,
                                       bool write_v2_headers)
{
  // NOTE: For dictionary encoding, parquet v2 recommends using PLAIN in dictionary page and
  // RLE_DICTIONARY in data page, but parquet v1 uses PLAIN_DICTIONARY in both dictionary and
  // data pages (actual encoding is identical).
  switch (page_type) {
    case PageType::DATA_PAGE: return use_dictionary ? Encoding::PLAIN_DICTIONARY : Encoding::PLAIN;
    case PageType::DATA_PAGE_V2:
      return physical_type == BOOLEAN ? Encoding::RLE
             : use_dictionary         ? Encoding::RLE_DICTIONARY
                                      : Encoding::PLAIN;
    case PageType::DICTIONARY_PAGE:
      return write_v2_headers ? Encoding::PLAIN : Encoding::PLAIN_DICTIONARY;
    default: CUDF_UNREACHABLE("unsupported page type");
  }
}

/**
 * @brief Generate level histogram for a page.
 *
 * For definition levels, the histogram values h(0)...h(max_def-1) represent nulls at
 * various levels of the hierarchy, and h(max_def) is the number of non-null values (num_valid).
 * If the leaf level is nullable, then num_leaf_values is h(max_def-1) + h(max_def),
 * and h(max_def-1) is num_leaf_values - num_valid. h(0) is derivable as num_values -
 * sum(h(1)..h(max_def)).
 *
 * For repetition levels, h(0) equals the number of rows. Here we can calculate
 * h(1)..h(max_rep-1), set h(0) directly, and then obtain h(max_rep) in the same way as
 * for the definition levels.
 *
 * @param hist Pointer to the histogram (size is max_level + 1)
 * @param s Page encode state
 * @param lvl_data Pointer to the global repetition or definition level data
 * @param lvl_end Last element of the histogram to encode (exclusive)
 */
template <int block_size, typename state_buf>
void __device__
generate_page_histogram(uint32_t* hist, state_buf const* s, uint8_t const* lvl_data, int lvl_end)
{
  using block_reduce = hipcub::BlockReduce<int, block_size>;
  __shared__ typename block_reduce::TempStorage temp_storage;

  auto const t                  = threadIdx.x;
  auto const page_first_val_idx = s->col.level_offsets[s->page.start_row];
  auto const col_last_val_idx   = s->col.level_offsets[s->col.num_rows];

  // h(0) is always derivable, so start at 1
  for (int lvl = 1; lvl < lvl_end; lvl++) {
    int nval_in_level = 0;
    for (int i = 0; i < s->page.num_values; i += block_size) {
      auto const lidx = i + t;
      auto const gidx = page_first_val_idx + lidx;
      if (lidx < s->page.num_values && gidx < col_last_val_idx && lvl_data[gidx] == lvl) {
        nval_in_level++;
      }
    }
    __syncthreads();
    auto const lvl_sum = block_reduce(temp_storage).Sum(nval_in_level);
    if (t == 0) { hist[lvl] = lvl_sum; }
  }
}

/**
 * @brief Generate definition level histogram for a block of values.
 *
 * This is used when the max repetition level is 0 (no lists) and the definition
 * level data is not calculated in advance for the entire column.
 *
 * @param hist Pointer to the histogram (size is max_def_level + 1)
 * @param s Page encode state
 * @param nrows Number of rows to process
 * @param rle_numvals Index (relative to start of page) of the first level value
 * @param maxlvl Last element of the histogram to encode (exclusive)
 */
template <int block_size>
void __device__ generate_def_level_histogram(uint32_t* hist,
                                             rle_page_enc_state_s const* s,
                                             uint32_t nrows,
                                             uint32_t rle_numvals,
                                             uint32_t maxlvl)
{
  using block_reduce = hipcub::BlockReduce<uint32_t, block_size>;
  __shared__ typename block_reduce::TempStorage temp_storage;
  auto const t = threadIdx.x;

  // Do a block sum for each level rather than each thread trying an atomicAdd.
  // This way is much faster.
  auto const mylvl = s->vals[rolling_index<rle_buffer_size>(rle_numvals + t)];
  // We can start at 1 because hist[0] can be derived.
  for (uint32_t lvl = 1; lvl < maxlvl; lvl++) {
    uint32_t const is_yes = t < nrows and mylvl == lvl;
    auto const lvl_sum    = block_reduce(temp_storage).Sum(is_yes);
    if (t == 0) { hist[lvl] += lvl_sum; }
    __syncthreads();
  }
}

// operator to use with warp_reduce. stolen from hipcub::Sum
struct BitwiseOr {
  /// Binary OR operator, returns <tt>a | b</tt>
  template <typename T>
  __host__ __device__ __forceinline__ T operator()(T const& a, T const& b) const
  {
    return a | b;
  }
};

// PT is the parquet physical type (INT32 or INT64).
// I is the column type from the input table.
template <Type PT, typename I>
__device__ uint8_t const* delta_encode(page_enc_state_s<0>* s, uint64_t* buffer, void* temp_space)
{
  using output_type = std::conditional_t<PT == INT32, int32_t, int64_t>;
  __shared__ delta_binary_packer<output_type> packer;

  auto const t = threadIdx.x;
  if (t == 0) {
    packer.init(s->cur, s->page.num_valid, reinterpret_cast<output_type*>(buffer), temp_space);
  }
  __syncthreads();

  // TODO(ets): in the plain encoder the scaling is a little different for INT32 than INT64.
  // might need to modify this if there's a big performance hit in the 32-bit case.
  int32_t const scale = s->col.ts_scale == 0 ? 1 : s->col.ts_scale;
  for (uint32_t cur_val_idx = 0; cur_val_idx < s->page.num_leaf_values;) {
    uint32_t const nvals = min(s->page.num_leaf_values - cur_val_idx, delta::block_size);

    size_type const val_idx_in_block = cur_val_idx + t;
    size_type const val_idx          = s->page_start_val + val_idx_in_block;

    bool const is_valid =
      (val_idx < s->col.leaf_column->size() && val_idx_in_block < s->page.num_leaf_values)
        ? s->col.leaf_column->is_valid(val_idx)
        : false;

    cur_val_idx += nvals;

    output_type v = is_valid ? s->col.leaf_column->element<I>(val_idx) : 0;
    if (scale < 0) {
      v /= -scale;
    } else {
      v *= scale;
    }
    packer.add_value(v, is_valid);
  }

  return packer.flush();
}

/**
 * @brief Sets `s->cur` to point to the start of encoded page data.
 *
 * For V1 headers, this will be immediately after the repetition and definition level data. For V2,
 * it will be at the next properly aligned location after the level data. The padding in V2 is
 * needed for compressors that require aligned input.
 */
template <typename state_type>
inline void __device__ set_page_data_start(state_type* s)
{
  s->cur = s->page.page_data + s->page.max_hdr_size;
  switch (s->page.page_type) {
    case PageType::DATA_PAGE:
      s->cur += s->page.level_bytes();
      if (s->col.num_def_level_bits() != 0) { s->cur += RLE_LENGTH_FIELD_LEN; }
      if (s->col.num_rep_level_bits() != 0) { s->cur += RLE_LENGTH_FIELD_LEN; }
      break;
    case PageType::DATA_PAGE_V2: s->cur += s->page.max_lvl_size; break;
  }
}

}  // anonymous namespace

// blockDim {512,1,1}
template <int block_size>
CUDF_KERNEL void __launch_bounds__(block_size)
  gpuInitRowGroupFragments(device_2dspan<PageFragment> frag,
                           device_span<parquet_column_device_view const> col_desc,
                           device_span<partition_info const> partitions,
                           device_span<int const> part_frag_offset,
                           uint32_t fragment_size)
{
  __shared__ __align__(16) frag_init_state_s state_g;

  frag_init_state_s* const s          = &state_g;
  auto const t                        = threadIdx.x;
  auto const num_fragments_per_column = frag.size().second;

  if (t == 0) { s->col = col_desc[blockIdx.x]; }
  __syncthreads();

  for (uint32_t frag_y = blockIdx.y; frag_y < num_fragments_per_column; frag_y += gridDim.y) {
    if (t == 0) {
      // Find which partition this fragment came from
      auto it =
        thrust::upper_bound(thrust::seq, part_frag_offset.begin(), part_frag_offset.end(), frag_y);
      int const p            = it - part_frag_offset.begin() - 1;
      int const part_end_row = partitions[p].start_row + partitions[p].num_rows;
      s->frag.start_row = (frag_y - part_frag_offset[p]) * fragment_size + partitions[p].start_row;
      s->frag.chunk     = frag[blockIdx.x][frag_y].chunk;
      init_frag_state(s, fragment_size, part_end_row);
    }
    __syncthreads();

    calculate_frag_size<block_size>(s, t);
    __syncthreads();
    if (t == 0) { frag[blockIdx.x][frag_y] = s->frag; }
  }
}

// blockDim {512,1,1}
template <int block_size>
CUDF_KERNEL void __launch_bounds__(block_size)
  gpuCalculatePageFragments(device_span<PageFragment> frag,
                            device_span<size_type const> column_frag_sizes)
{
  __shared__ __align__(16) frag_init_state_s state_g;

  EncColumnChunk* const ck_g = frag[blockIdx.x].chunk;
  frag_init_state_s* const s = &state_g;
  uint32_t const t           = threadIdx.x;
  auto const fragment_size   = column_frag_sizes[ck_g->col_desc_id];

  if (t == 0) { s->col = *ck_g->col_desc; }
  __syncthreads();

  if (t == 0) {
    int const part_end_row = ck_g->start_row + ck_g->num_rows;
    s->frag.start_row      = ck_g->start_row + (blockIdx.x - ck_g->first_fragment) * fragment_size;
    s->frag.chunk          = ck_g;
    init_frag_state(s, fragment_size, part_end_row);
  }
  __syncthreads();

  calculate_frag_size<block_size>(s, t);
  if (t == 0) { frag[blockIdx.x] = s->frag; }
}

// blockDim {128,1,1}
CUDF_KERNEL void __launch_bounds__(128)
  gpuInitFragmentStats(device_span<statistics_group> groups,
                       device_span<PageFragment const> fragments)
{
  uint32_t const lane_id = threadIdx.x & WARP_MASK;
  uint32_t const frag_id = blockIdx.x * 4 + (threadIdx.x / cudf::detail::warp_size);
  if (frag_id < fragments.size()) {
    if (lane_id == 0) {
      statistics_group g;
      auto* const ck_g = fragments[frag_id].chunk;
      g.col            = ck_g->col_desc;
      g.start_row      = fragments[frag_id].start_value_idx;
      g.num_rows       = fragments[frag_id].num_leaf_values;
      g.non_leaf_nulls = fragments[frag_id].num_values - g.num_rows;
      groups[frag_id]  = g;
    }
  }
}

// given a column chunk, determine which data encoding to use
__device__ encode_kernel_mask data_encoding_for_col(EncColumnChunk const* chunk,
                                                    parquet_column_device_view const* col_desc,
                                                    bool write_v2_headers)
{
  // first check for dictionary (boolean always uses dict encoder)
  if (chunk->use_dictionary or col_desc->physical_type == BOOLEAN) {
    return encode_kernel_mask::DICTIONARY;
  }

  // next check for user requested encoding, but skip if user requested dictionary encoding
  // (if we could use the requested dict encoding, we'd have returned above)
  if (col_desc->requested_encoding != column_encoding::USE_DEFAULT and
      col_desc->requested_encoding != column_encoding::DICTIONARY) {
    switch (col_desc->requested_encoding) {
      case column_encoding::PLAIN: return encode_kernel_mask::PLAIN;
      case column_encoding::DELTA_BINARY_PACKED: return encode_kernel_mask::DELTA_BINARY;
      case column_encoding::DELTA_LENGTH_BYTE_ARRAY: return encode_kernel_mask::DELTA_LENGTH_BA;
      case column_encoding::DELTA_BYTE_ARRAY: return encode_kernel_mask::DELTA_BYTE_ARRAY;
    }
  }

  // Select a fallback encoding. For V1, we always choose PLAIN. For V2 we'll use
  // DELTA_BINARY_PACKED for INT32 and INT64, and DELTA_LENGTH_BYTE_ARRAY for
  // BYTE_ARRAY. Everything else will still fall back to PLAIN.
  if (write_v2_headers) {
    switch (col_desc->physical_type) {
      case INT32:
      case INT64: return encode_kernel_mask::DELTA_BINARY;
      case BYTE_ARRAY: return encode_kernel_mask::DELTA_LENGTH_BA;
    }
  }

  return encode_kernel_mask::PLAIN;
}

__device__ size_t delta_data_len(Type physical_type,
                                 cudf::type_id type_id,
                                 uint32_t num_values,
                                 size_t page_size,
                                 encode_kernel_mask encoding)
{
  auto const dtype_len_out = physical_type_len(physical_type, type_id);
  auto const dtype_len     = [&]() -> uint32_t {
    if (physical_type == INT32) { return int32_logical_len(type_id); }
    if (physical_type == INT96) { return sizeof(int64_t); }
    return dtype_len_out;
  }();

  auto const vals_per_block = delta::block_size;
  size_t const num_blocks   = util::div_rounding_up_unsafe(num_values, vals_per_block);
  // need max dtype_len + 1 bytes for min_delta (because we only encode 7 bits per byte)
  // one byte per mini block for the bitwidth
  auto const mini_block_header_size = dtype_len + 1 + delta::num_mini_blocks;
  // each encoded value can be at most sizeof(type) * 8 + 1 bits
  auto const max_bits = dtype_len * 8 + 1;
  // each data block will then be max_bits * values per block. vals_per_block is guaranteed to be
  // divisible by 128 (via static assert on delta::block_size), but do safe division anyway.
  auto const bytes_per_block = cudf::util::div_rounding_up_unsafe(max_bits * vals_per_block, 8);
  auto const block_size      = mini_block_header_size + bytes_per_block;
  // the number of DELTA_BINARY_PACKED blocks to encode
  auto const num_dbp_blocks = encoding == encode_kernel_mask::DELTA_BYTE_ARRAY ? 2 : 1;

  // delta header is 2 bytes for the block_size, 1 byte for number of mini-blocks,
  // max 5 bytes for number of values, and max dtype_len + 1 for first value.
  // TODO: if we ever allow configurable block sizes then this calculation will need to be
  // modified.
  auto const header_size = 2 + 1 + 5 + dtype_len + 1;

  // The above is just a size estimate for a DELTA_BINARY_PACKED data page. For BYTE_ARRAY
  // data we also need to add size of the char data. `page_size` that is passed in is the
  // plain encoded size (i.e. num_values * sizeof(size_type) + char_data_len), so the char
  // data len is `page_size` minus the first term. For FIXED_LEN_BYTE_ARRAY there are no
  // lengths, so just use `page_size`.
  // `num_dbp_blocks` takes into account the two delta binary blocks for DELTA_BYTE_ARRAY.
  size_t char_data_len = 0;
  if (physical_type == BYTE_ARRAY) {
    char_data_len = page_size - num_values * sizeof(size_type);
  } else if (physical_type == FIXED_LEN_BYTE_ARRAY) {
    char_data_len = page_size;
  }

  return header_size + num_blocks * num_dbp_blocks * block_size + char_data_len;
}

// blockDim {128,1,1}
CUDF_KERNEL void __launch_bounds__(128)
  gpuInitPages(device_2dspan<EncColumnChunk> chunks,
               device_span<EncPage> pages,
               device_span<size_type> page_sizes,
               device_span<size_type> comp_page_sizes,
               device_span<parquet_column_device_view const> col_desc,
               statistics_merge_group* page_grstats,
               statistics_merge_group* chunk_grstats,
               int32_t num_columns,
               size_t max_page_size_bytes,
               size_type max_page_size_rows,
               uint32_t page_align,
               bool write_v2_headers)
{
  // TODO: All writing seems to be done by thread 0. Could be replaced by thrust foreach
  __shared__ __align__(8) parquet_column_device_view col_g;
  __shared__ __align__(8) EncColumnChunk ck_g;
  __shared__ __align__(8) PageFragment frag_g;
  __shared__ __align__(8) EncPage page_g;
  __shared__ __align__(8) statistics_merge_group pagestats_g;

  uint32_t const t          = threadIdx.x;
  auto const data_page_type = write_v2_headers ? PageType::DATA_PAGE_V2 : PageType::DATA_PAGE;

  // Max page header size excluding statistics
  auto const max_data_page_hdr_size = write_v2_headers ? MAX_V2_HDR_SIZE : MAX_V1_HDR_SIZE;

  if (t == 0) {
    col_g  = col_desc[blockIdx.x];
    ck_g   = chunks[blockIdx.y][blockIdx.x];
    page_g = {};
  }
  __syncthreads();

  // if writing delta encoded values, we're going to need to know the data length to get a guess
  // at the worst case number of bytes needed to encode.
  auto const physical_type = col_g.physical_type;
  auto const type_id       = col_g.leaf_column->type().id();

  // figure out kernel encoding to use for data pages
  auto const column_data_encoding = data_encoding_for_col(&ck_g, &col_g, write_v2_headers);
  auto const is_use_delta         = column_data_encoding == encode_kernel_mask::DELTA_BINARY or
                            column_data_encoding == encode_kernel_mask::DELTA_LENGTH_BA or
                            column_data_encoding == encode_kernel_mask::DELTA_BYTE_ARRAY;

  if (t < 32) {
    uint32_t fragments_in_chunk  = 0;
    uint32_t rows_in_page        = 0;
    uint32_t values_in_page      = 0;
    uint32_t leaf_values_in_page = 0;
    uint32_t num_valid           = 0;
    size_t page_size             = 0;
    size_t var_bytes_size        = 0;
    uint32_t num_pages           = 0;
    uint32_t num_rows            = 0;
    uint32_t page_start          = 0;
    uint32_t page_offset         = ck_g.ck_stat_size;
    uint32_t num_dict_entries    = 0;
    uint32_t comp_page_offset    = ck_g.ck_stat_size;
    uint32_t page_headers_size   = 0;
    uint32_t max_page_data_size  = 0;
    uint32_t cur_row             = ck_g.start_row;
    uint32_t ck_max_stats_len    = 0;
    uint32_t max_stats_len       = 0;

    if (!t) {
      pagestats_g.col_dtype   = col_g.leaf_column->type();
      pagestats_g.stats_dtype = col_g.stats_dtype;
      pagestats_g.start_chunk = ck_g.first_fragment;
      pagestats_g.num_chunks  = 0;
    }
    if (ck_g.use_dictionary) {
      if (!t) {
        page_g.page_data       = ck_g.uncompressed_bfr + page_offset;
        page_g.compressed_data = ck_g.compressed_bfr + comp_page_offset;
        page_g.num_fragments   = 0;
        page_g.page_type       = PageType::DICTIONARY_PAGE;
        page_g.chunk           = &chunks[blockIdx.y][blockIdx.x];
        page_g.chunk_id        = blockIdx.y * num_columns + blockIdx.x;
        page_g.hdr_size        = 0;
        page_g.def_lvl_bytes   = 0;
        page_g.rep_lvl_bytes   = 0;
        page_g.max_lvl_size    = 0;
        page_g.comp_data_size  = 0;
        page_g.max_hdr_size    = MAX_V1_HDR_SIZE;
        page_g.max_data_size   = ck_g.uniq_data_size;
        page_g.data_size       = ck_g.uniq_data_size;
        page_g.start_row       = cur_row;
        page_g.num_rows        = ck_g.num_dict_entries;
        page_g.num_leaf_values = ck_g.num_dict_entries;
        page_g.num_values      = ck_g.num_dict_entries;  // TODO: shouldn't matter for dict page
        page_offset +=
          util::round_up_unsafe(page_g.max_hdr_size + page_g.max_data_size, page_align);
        if (not comp_page_sizes.empty()) {
          comp_page_offset += page_g.max_hdr_size + comp_page_sizes[ck_g.first_page];
        }
        page_headers_size += page_g.max_hdr_size;
        max_page_data_size = max(max_page_data_size, page_g.max_data_size);
      }
      __syncwarp();
      if (t == 0) {
        if (not pages.empty()) {
          page_g.kernel_mask     = encode_kernel_mask::PLAIN;
          pages[ck_g.first_page] = page_g;
        }
        if (not page_sizes.empty()) { page_sizes[ck_g.first_page] = page_g.max_data_size; }
        if (page_grstats) { page_grstats[ck_g.first_page] = pagestats_g; }
      }
      num_pages = 1;
    }
    __syncwarp();

    // page padding needed for RLE encoded boolean data
    auto const rle_pad =
      write_v2_headers && col_g.physical_type == BOOLEAN ? RLE_LENGTH_FIELD_LEN : 0;

    // This loop goes over one page fragment at a time and adds it to page.
    // When page size crosses a particular limit, then it moves on to the next page and then next
    // page fragment gets added to that one.

    // This doesn't actually deal with data. It's agnostic. It only cares about number of rows and
    // page size.
    do {
      uint32_t minmax_len = 0;
      __syncwarp();
      if (num_rows < ck_g.num_rows) {
        if (t == 0) { frag_g = ck_g.fragments[fragments_in_chunk]; }
        if (!t && ck_g.stats) {
          if (col_g.stats_dtype == dtype_string) {
            minmax_len = max(ck_g.stats[fragments_in_chunk].min_value.str_val.length,
                             ck_g.stats[fragments_in_chunk].max_value.str_val.length);
          } else if (col_g.stats_dtype == dtype_byte_array) {
            minmax_len = max(ck_g.stats[fragments_in_chunk].min_value.byte_val.length,
                             ck_g.stats[fragments_in_chunk].max_value.byte_val.length);
          }
        }
      } else if (!t) {
        frag_g.fragment_data_size = 0;
        frag_g.num_rows           = 0;
      }
      __syncwarp();
      uint32_t fragment_data_size =
        (ck_g.use_dictionary)
          ? frag_g.num_leaf_values * util::div_rounding_up_unsafe(ck_g.dict_rle_bits, 8)
          : frag_g.fragment_data_size;

      // page fragment size must fit in a 32-bit signed integer
      if (fragment_data_size > std::numeric_limits<int32_t>::max()) {
        CUDF_UNREACHABLE("page fragment size exceeds maximum for i32");
      }

      // TODO (dm): this convoluted logic to limit page size needs refactoring
      size_t this_max_page_size = (values_in_page * 2 >= ck_g.num_values)   ? 256 * 1024
                                  : (values_in_page * 3 >= ck_g.num_values) ? 384 * 1024
                                                                            : 512 * 1024;

      // override this_max_page_size if the requested size is smaller
      this_max_page_size = min(this_max_page_size, max_page_size_bytes);

      // subtract size of rep and def level vectors and RLE length field
      auto num_vals      = values_in_page + frag_g.num_values;
      this_max_page_size = underflow_safe_subtract(
        this_max_page_size,
        max_RLE_page_size(col_g.num_def_level_bits(), num_vals) +
          max_RLE_page_size(col_g.num_rep_level_bits(), num_vals) + rle_pad);

      // checks to see when we need to close the current page and start a new one
      auto const is_last_chunk          = num_rows >= ck_g.num_rows;
      auto const is_page_bytes_exceeded = page_size + fragment_data_size > this_max_page_size;
      auto const is_page_rows_exceeded  = rows_in_page + frag_g.num_rows > max_page_size_rows;
      // only check for limit overflow if there's already at least one fragment for this page
      auto const is_page_too_big =
        values_in_page > 0 && (is_page_bytes_exceeded || is_page_rows_exceeded);

      if (is_last_chunk || is_page_too_big) {
        if (ck_g.use_dictionary) {
          // Additional byte to store entry bit width
          page_size = 1 + max_RLE_page_size(ck_g.dict_rle_bits, values_in_page);
        }
        if (!t) {
          page_g.num_fragments  = fragments_in_chunk - page_start;
          page_g.chunk          = &chunks[blockIdx.y][blockIdx.x];
          page_g.chunk_id       = blockIdx.y * num_columns + blockIdx.x;
          page_g.page_type      = data_page_type;
          page_g.hdr_size       = 0;
          page_g.def_lvl_bytes  = 0;
          page_g.rep_lvl_bytes  = 0;
          page_g.max_lvl_size   = 0;
          page_g.data_size      = 0;
          page_g.comp_data_size = 0;
          page_g.max_hdr_size   = max_data_page_hdr_size;  // Max size excluding statistics
          if (ck_g.stats) {
            uint32_t stats_hdr_len = 16;
            if (col_g.stats_dtype == dtype_string || col_g.stats_dtype == dtype_byte_array) {
              stats_hdr_len += 5 * 3 + 2 * max_stats_len;
            } else {
              stats_hdr_len += ((col_g.stats_dtype >= dtype_int64) ? 10 : 5) * 3;
            }
            page_g.max_hdr_size += stats_hdr_len;
          }
          page_g.max_hdr_size = util::round_up_unsafe(page_g.max_hdr_size, page_align);
          page_g.page_data    = ck_g.uncompressed_bfr + page_offset;
          if (not comp_page_sizes.empty()) {
            page_g.compressed_data = ck_g.compressed_bfr + comp_page_offset;
          }
          page_g.start_row          = cur_row;
          page_g.num_rows           = rows_in_page;
          page_g.num_leaf_values    = leaf_values_in_page;
          page_g.num_values         = values_in_page;
          page_g.num_valid          = num_valid;
          auto const def_level_size = max_RLE_page_size(col_g.num_def_level_bits(), values_in_page);
          auto const rep_level_size = max_RLE_page_size(col_g.num_rep_level_bits(), values_in_page);
          if (write_v2_headers) {
            page_g.max_lvl_size =
              util::round_up_unsafe(def_level_size + rep_level_size, page_align);
          }
          // get a different bound if using delta encoding
          if (is_use_delta) {
            auto const delta_len = delta_data_len(
              physical_type, type_id, page_g.num_leaf_values, page_size, column_data_encoding);
            page_size = max(page_size, delta_len);
          }
          auto const max_data_size =
            page_size + rle_pad +
            (write_v2_headers ? page_g.max_lvl_size : def_level_size + rep_level_size);
          // page size must fit in 32-bit signed integer
          if (max_data_size > std::numeric_limits<int32_t>::max()) {
            CUDF_UNREACHABLE("page size exceeds maximum for i32");
          }
          // if byte_array then save the variable bytes size
          if (ck_g.col_desc->physical_type == BYTE_ARRAY) {
            // Page size is the sum of frag sizes, and frag sizes for strings includes the
            // 4-byte length indicator, so subtract that.
            page_g.var_bytes_size = var_bytes_size;
          }

          page_g.kernel_mask      = column_data_encoding;
          page_g.max_data_size    = static_cast<uint32_t>(max_data_size);
          pagestats_g.start_chunk = ck_g.first_fragment + page_start;
          pagestats_g.num_chunks  = page_g.num_fragments;
          page_offset +=
            util::round_up_unsafe(page_g.max_hdr_size + page_g.max_data_size, page_align);
          // if encoding delta_byte_array, need to allocate some space for scratch data.
          // if there are leaf nulls, we need space for a mapping array:
          //   sizeof(size_type) * num_leaf_values
          // we always need prefix lengths: sizeof(size_type) * num_valid
          if (page_g.kernel_mask == encode_kernel_mask::DELTA_BYTE_ARRAY) {
            // scratch needs to be aligned to a size_type boundary
            auto const pg_end = reinterpret_cast<uintptr_t>(ck_g.uncompressed_bfr + page_offset);
            auto scratch      = util::round_up_unsafe(pg_end, sizeof(size_type));
            if (page_g.num_valid != page_g.num_leaf_values) {
              scratch += sizeof(size_type) * page_g.num_leaf_values;
            }
            scratch += sizeof(size_type) * page_g.num_valid;
            page_offset =
              thrust::distance(ck_g.uncompressed_bfr, reinterpret_cast<uint8_t*>(scratch));
          }
          if (not comp_page_sizes.empty()) {
            // V2 does not include level data in compressed size estimate
            comp_page_offset += page_g.max_hdr_size + page_g.max_lvl_size +
                                comp_page_sizes[ck_g.first_page + num_pages];
          }
          page_headers_size += page_g.max_hdr_size;
          max_page_data_size = max(max_page_data_size, page_g.max_data_size);
          cur_row += rows_in_page;
          ck_max_stats_len = max(ck_max_stats_len, max_stats_len);
        }
        __syncwarp();
        if (t == 0) {
          if (not pages.empty()) {
            // need space for the chunk histograms plus data page histograms
            auto const num_histograms = num_pages - ck_g.num_dict_pages();
            if (ck_g.def_histogram_data != nullptr && col_g.max_def_level > 0) {
              page_g.def_histogram =
                ck_g.def_histogram_data + num_histograms * (col_g.max_def_level + 1);
            }
            if (ck_g.rep_histogram_data != nullptr && col_g.max_rep_level > 0) {
              page_g.rep_histogram =
                ck_g.rep_histogram_data + num_histograms * (col_g.max_rep_level + 1);
            }
            pages[ck_g.first_page + num_pages] = page_g;
          }
          // page_sizes should be the number of bytes to be compressed, so don't include level
          // data for V2.
          if (not page_sizes.empty()) {
            page_sizes[ck_g.first_page + num_pages] = page_g.max_data_size - page_g.max_lvl_size;
          }
          if (page_grstats) { page_grstats[ck_g.first_page + num_pages] = pagestats_g; }
        }

        num_pages++;
        page_size           = 0;
        var_bytes_size      = 0;
        rows_in_page        = 0;
        values_in_page      = 0;
        leaf_values_in_page = 0;
        num_valid           = 0;
        page_start          = fragments_in_chunk;
        max_stats_len       = 0;
      }
      max_stats_len = max(max_stats_len, minmax_len);
      num_dict_entries += frag_g.num_dict_vals;
      page_size += fragment_data_size;
      // fragment_data_size includes the length indicator...remove it
      var_bytes_size += frag_g.fragment_data_size - frag_g.num_valid * sizeof(size_type);
      rows_in_page += frag_g.num_rows;
      values_in_page += frag_g.num_values;
      leaf_values_in_page += frag_g.num_leaf_values;
      num_valid += frag_g.num_valid;
      num_rows += frag_g.num_rows;
      fragments_in_chunk++;
    } while (frag_g.num_rows != 0);
    __syncwarp();
    if (!t) {
      if (ck_g.ck_stat_size == 0 && ck_g.stats) {
        uint32_t ck_stat_size = util::round_up_unsafe(48 + 2 * ck_max_stats_len, page_align);
        page_offset += ck_stat_size;
        comp_page_offset += ck_stat_size;
        ck_g.ck_stat_size = ck_stat_size;
      }
      ck_g.num_pages          = num_pages;
      ck_g.bfr_size           = page_offset;
      ck_g.page_headers_size  = page_headers_size;
      ck_g.max_page_data_size = max_page_data_size;
      if (not comp_page_sizes.empty()) { ck_g.compressed_size = comp_page_offset; }
      pagestats_g.start_chunk = ck_g.first_page + ck_g.use_dictionary;  // Exclude dictionary
      pagestats_g.num_chunks  = num_pages - ck_g.use_dictionary;
    }
  }
  __syncthreads();
  if (t == 0) {
    if (not pages.empty()) ck_g.pages = &pages[ck_g.first_page];
    chunks[blockIdx.y][blockIdx.x] = ck_g;
    if (chunk_grstats) chunk_grstats[blockIdx.y * num_columns + blockIdx.x] = pagestats_g;
  }
}

/**
 * @brief Mask table representing how many consecutive repeats are needed to code a repeat run
 *[nbits-1]
 */
static __device__ __constant__ uint32_t kRleRunMask[24] = {
  0x00ff'ffff, 0x0fff, 0x00ff, 0x3f, 0x0f, 0x0f, 0x7, 0x7, 0x3, 0x3, 0x3, 0x3,
  0x1,         0x1,    0x1,    0x1,  0x1,  0x1,  0x1, 0x1, 0x1, 0x1, 0x1, 0x1};

/**
 * @brief Variable-length encode an integer
 */
inline __device__ uint8_t* VlqEncode(uint8_t* p, uint32_t v)
{
  while (v > 0x7f) {
    *p++ = (v | 0x80);
    v >>= 7;
  }
  *p++ = v;
  return p;
}

/**
 * @brief Pack literal values in output bitstream (1,2,3,4,5,6,8,10,12,16,20 or 24 bits per value)
 */
inline __device__ void PackLiteralsShuffle(
  uint8_t* dst, uint32_t v, uint32_t count, uint32_t w, uint32_t t)
{
  constexpr uint32_t MASK2T = 1;  // mask for 2 thread leader
  constexpr uint32_t MASK4T = 3;  // mask for 4 thread leader
  constexpr uint32_t MASK8T = 7;  // mask for 8 thread leader
  uint64_t v64;

  if (t > (count | 0x1f)) { return; }

  switch (w) {
    case 1:
      v |= shuffle_xor(v, 1) << 1;  // grab bit 1 from neighbor
      v |= shuffle_xor(v, 2) << 2;  // grab bits 2-3 from 2 lanes over
      v |= shuffle_xor(v, 4) << 4;  // grab bits 4-7 from 4 lanes over
      // sub-warp leader writes the combined bits
      if (t < count && !(t & MASK8T)) { dst[(t * w) >> 3] = v; }
      return;
    case 2:
      v |= shuffle_xor(v, 1) << 2;
      v |= shuffle_xor(v, 2) << 4;
      if (t < count && !(t & MASK4T)) { dst[(t * w) >> 3] = v; }
      return;
    case 3:
      v |= shuffle_xor(v, 1) << 3;
      v |= shuffle_xor(v, 2) << 6;
      v |= shuffle_xor(v, 4) << 12;
      if (t < count && !(t & MASK8T)) {
        dst[(t >> 3) * 3 + 0] = v;
        dst[(t >> 3) * 3 + 1] = v >> 8;
        dst[(t >> 3) * 3 + 2] = v >> 16;
      }
      return;
    case 4:
      v |= shuffle_xor(v, 1) << 4;
      if (t < count && !(t & MASK2T)) { dst[(t * w) >> 3] = v; }
      return;
    case 5:
      v |= shuffle_xor(v, 1) << 5;
      v |= shuffle_xor(v, 2) << 10;
      v64 = static_cast<uint64_t>(shuffle_xor(v, 4)) << 20 | v;
      if (t < count && !(t & MASK8T)) {
        dst[(t >> 3) * 5 + 0] = v64;
        dst[(t >> 3) * 5 + 1] = v64 >> 8;
        dst[(t >> 3) * 5 + 2] = v64 >> 16;
        dst[(t >> 3) * 5 + 3] = v64 >> 24;
        dst[(t >> 3) * 5 + 4] = v64 >> 32;
      }
      return;
    case 6:
      v |= shuffle_xor(v, 1) << 6;
      v |= shuffle_xor(v, 2) << 12;
      if (t < count && !(t & MASK4T)) {
        dst[(t >> 2) * 3 + 0] = v;
        dst[(t >> 2) * 3 + 1] = v >> 8;
        dst[(t >> 2) * 3 + 2] = v >> 16;
      }
      return;
    case 8:
      if (t < count) { dst[t] = v; }
      return;
    case 10:
      v |= shuffle_xor(v, 1) << 10;
      v64 = static_cast<uint64_t>(shuffle_xor(v, 2)) << 20 | v;
      if (t < count && !(t & MASK4T)) {
        dst[(t >> 2) * 5 + 0] = v64;
        dst[(t >> 2) * 5 + 1] = v64 >> 8;
        dst[(t >> 2) * 5 + 2] = v64 >> 16;
        dst[(t >> 2) * 5 + 3] = v64 >> 24;
        dst[(t >> 2) * 5 + 4] = v64 >> 32;
      }
      return;
    case 12:
      v |= shuffle_xor(v, 1) << 12;
      if (t < count && !(t & MASK2T)) {
        dst[(t >> 1) * 3 + 0] = v;
        dst[(t >> 1) * 3 + 1] = v >> 8;
        dst[(t >> 1) * 3 + 2] = v >> 16;
      }
      return;
    case 16:
      if (t < count) {
        dst[t * 2 + 0] = v;
        dst[t * 2 + 1] = v >> 8;
      }
      return;
    case 20:
      v64 = static_cast<uint64_t>(shuffle_xor(v, 1)) << 20 | v;
      if (t < count && !(t & MASK2T)) {
        dst[(t >> 1) * 5 + 0] = v64;
        dst[(t >> 1) * 5 + 1] = v64 >> 8;
        dst[(t >> 1) * 5 + 2] = v64 >> 16;
        dst[(t >> 1) * 5 + 3] = v64 >> 24;
        dst[(t >> 1) * 5 + 4] = v64 >> 32;
      }
      return;
    case 24:
      if (t < count) {
        dst[t * 3 + 0] = v;
        dst[t * 3 + 1] = v >> 8;
        dst[t * 3 + 2] = v >> 16;
      }
      return;

    default: CUDF_UNREACHABLE("Unsupported bit width");
  }
}

/**
 * @brief Pack literals of arbitrary bit-length in output bitstream.
 */
inline __device__ void PackLiteralsRoundRobin(
  uint8_t* dst, uint32_t v, uint32_t count, uint32_t w, uint32_t t)
{
  // Scratch space to temporarily write to. Needed because we will use atomics to write 32 bit
  // words but the destination mem may not be a multiple of 4 bytes.
  // TODO (dm): This assumes blockdim = 128. Reduce magic numbers.
  constexpr uint32_t NUM_THREADS  = 128;  // this needs to match gpuEncodePages block_size parameter
  constexpr uint32_t NUM_BYTES    = (NUM_THREADS * MAX_DICT_BITS) >> 3;
  constexpr uint32_t SCRATCH_SIZE = NUM_BYTES / sizeof(uint32_t);
  __shared__ uint32_t scratch[SCRATCH_SIZE];
  for (uint32_t i = t; i < SCRATCH_SIZE; i += NUM_THREADS) {
    scratch[i] = 0;
  }
  __syncthreads();

  if (t <= count) {
    // shift symbol left by up to 31 bits
    uint64_t v64 = v;
    v64 <<= (t * w) & 0x1f;

    // Copy 64 bit word into two 32 bit words while following C++ strict aliasing rules.
    uint32_t v32[2];
    memcpy(&v32, &v64, sizeof(uint64_t));

    // Atomically write result to scratch
    if (v32[0]) { atomicOr(scratch + ((t * w) >> 5), v32[0]); }
    if (v32[1]) { atomicOr(scratch + ((t * w) >> 5) + 1, v32[1]); }
  }
  __syncthreads();

  // Copy scratch data to final destination
  auto available_bytes = (count * w + 7) / 8;

  auto scratch_bytes = reinterpret_cast<char*>(&scratch[0]);
  for (uint32_t i = t; i < available_bytes; i += NUM_THREADS) {
    dst[i] = scratch_bytes[i];
  }
  __syncthreads();
}

/**
 * @brief Pack literal values in output bitstream
 */
inline __device__ void PackLiterals(
  uint8_t* dst, uint32_t v, uint32_t count, uint32_t w, uint32_t t)
{
  if (w > 24) { CUDF_UNREACHABLE("Unsupported bit width"); }
  switch (w) {
    case 1:
    case 2:
    case 3:
    case 4:
    case 5:
    case 6:
    case 8:
    case 10:
    case 12:
    case 16:
    case 20:
    case 24:
      // bit widths that lie on easy boundaries can be handled either directly
      // (8, 16, 24) or through fast shuffle operations.
      PackLiteralsShuffle(dst, v, count, w, t);
      return;
    default:
      // bit packing that uses atomics, but can handle arbitrary bit widths up to 24.
      PackLiteralsRoundRobin(dst, v, count, w, t);
  }
}

/**
 * @brief RLE encoder
 *
 * @param[in,out] s Page encode state
 * @param[in] numvals Total count of input values
 * @param[in] nbits number of bits per symbol (1..16)
 * @param[in] flush nonzero if last batch in block
 * @param[in] t thread id (0..127)
 */
static __device__ void RleEncode(
  rle_page_enc_state_s* s, uint32_t numvals, uint32_t nbits, uint32_t flush, uint32_t t)
{
  using cudf::detail::warp_size;
  auto const lane_id = t % warp_size;
  auto const warp_id = t / warp_size;

  uint32_t rle_pos = s->rle_pos;
  uint32_t rle_run = s->rle_run;

  while (rle_pos < numvals || (flush && rle_run)) {
    uint32_t pos = rle_pos + t;
    if (rle_run > 0 && !(rle_run & 1)) {
      // Currently in a long repeat run
      uint32_t mask = ballot(pos < numvals && s->vals[rolling_idx(pos)] == s->run_val);
      uint32_t rle_rpt_count, max_rpt_count;
      if (lane_id == 0) { s->rpt_map[warp_id] = mask; }
      __syncthreads();
      if (t < warp_size) {
        uint32_t c32 = ballot(t >= 4 || s->rpt_map[t] != 0xffff'ffffu);
        if (t == 0) {
          uint32_t last_idx = __ffs(c32) - 1;
          s->rle_rpt_count =
            last_idx * warp_size + ((last_idx < 4) ? __ffs(~s->rpt_map[last_idx]) - 1 : 0);
        }
      }
      __syncthreads();
      max_rpt_count = min(numvals - rle_pos, encode_block_size);
      rle_rpt_count = s->rle_rpt_count;
      rle_run += rle_rpt_count << 1;
      rle_pos += rle_rpt_count;
      if (rle_rpt_count < max_rpt_count || (flush && rle_pos == numvals)) {
        if (t == 0) {
          uint32_t const run_val = s->run_val;
          uint8_t* dst           = VlqEncode(s->rle_out, rle_run);
          *dst++                 = run_val;
          if (nbits > 8) { *dst++ = run_val >> 8; }
          if (nbits > 16) { *dst++ = run_val >> 16; }
          s->rle_out = dst;
        }
        rle_run = 0;
      }
    } else {
      // New run or in a literal run
      uint32_t v0      = s->vals[rolling_idx(pos)];
      uint32_t v1      = s->vals[rolling_idx(pos + 1)];
      uint32_t mask    = ballot(pos + 1 < numvals && v0 == v1);
      uint32_t maxvals = min(numvals - rle_pos, encode_block_size);
      uint32_t rle_lit_count, rle_rpt_count;
      if (lane_id == 0) { s->rpt_map[warp_id] = mask; }
      __syncthreads();
      if (t < warp_size) {
        // Repeat run can only start on a multiple of 8 values
        uint32_t idx8        = (t * 8) / warp_size;
        uint32_t pos8        = (t * 8) % warp_size;
        uint32_t m0          = (idx8 < 4) ? s->rpt_map[idx8] : 0;
        uint32_t m1          = (idx8 < 3) ? s->rpt_map[idx8 + 1] : 0;
        uint32_t needed_mask = kRleRunMask[nbits - 1];
        mask                 = ballot((__funnelshift_r(m0, m1, pos8) & needed_mask) == needed_mask);
        if (!t) {
          uint32_t rle_run_start = (mask != 0) ? min((__ffs(mask) - 1) * 8, maxvals) : maxvals;
          uint32_t rpt_len       = 0;
          if (rle_run_start < maxvals) {
            uint32_t idx_cur = rle_run_start / warp_size;
            uint32_t idx_ofs = rle_run_start % warp_size;
            while (idx_cur < 4) {
              m0   = (idx_cur < 4) ? s->rpt_map[idx_cur] : 0;
              m1   = (idx_cur < 3) ? s->rpt_map[idx_cur + 1] : 0;
              mask = ~__funnelshift_r(m0, m1, idx_ofs);
              if (mask != 0) {
                rpt_len += __ffs(mask) - 1;
                break;
              }
              rpt_len += warp_size;
              idx_cur++;
            }
          }
          s->rle_lit_count = rle_run_start;
          s->rle_rpt_count = min(rpt_len, maxvals - rle_run_start);
        }
      }
      __syncthreads();
      rle_lit_count = s->rle_lit_count;
      rle_rpt_count = s->rle_rpt_count;
      if (rle_lit_count != 0 || (rle_run != 0 && rle_rpt_count != 0)) {
        uint32_t lit_div8;
        bool need_more_data = false;
        if (!flush && rle_pos + rle_lit_count == numvals) {
          // Wait for more data
          rle_lit_count -= min(rle_lit_count, 24);
          need_more_data = true;
        }
        if (rle_lit_count != 0) {
          lit_div8 = (rle_lit_count + ((flush && rle_pos + rle_lit_count == numvals) ? 7 : 0)) >> 3;
          if (rle_run + lit_div8 * 2 > 0x7f) {
            lit_div8      = 0x3f - (rle_run >> 1);  // Limit to fixed 1-byte header (504 literals)
            rle_rpt_count = 0;                      // Defer repeat run
          }
          if (lit_div8 != 0) {
            uint8_t* dst = s->rle_out + 1 + (rle_run >> 1) * nbits;
            PackLiterals(dst, (rle_pos + t < numvals) ? v0 : 0, lit_div8 * 8, nbits, t);
            rle_run = (rle_run + lit_div8 * 2) | 1;
            rle_pos = min(rle_pos + lit_div8 * 8, numvals);
          }
        }
        if (rle_run >= ((rle_rpt_count != 0 || (flush && rle_pos == numvals)) ? 0x03 : 0x7f)) {
          __syncthreads();
          // Complete literal run
          if (!t) {
            uint8_t* dst = s->rle_out;
            dst[0]       = rle_run;  // At most 0x7f
            dst += 1 + nbits * (rle_run >> 1);
            s->rle_out = dst;
          }
          rle_run = 0;
        }
        if (need_more_data) { break; }
      }
      // Start a repeat run
      if (rle_rpt_count != 0) {
        if (t == s->rle_lit_count) { s->run_val = v0; }
        rle_run = rle_rpt_count * 2;
        rle_pos += rle_rpt_count;
        if (rle_pos + 1 == numvals && !flush) { break; }
      }
    }
    __syncthreads();
  }
  __syncthreads();
  if (!t) {
    s->rle_run     = rle_run;
    s->rle_pos     = rle_pos;
    s->rle_numvals = numvals;
  }
}

/**
 * @brief PLAIN bool encoder
 *
 * @param[in,out] s Page encode state
 * @param[in] numvals Total count of input values
 * @param[in] flush nonzero if last batch in block
 * @param[in] t thread id (0..127)
 */
static __device__ void PlainBoolEncode(rle_page_enc_state_s* s,
                                       uint32_t numvals,
                                       uint32_t flush,
                                       uint32_t t)
{
  uint32_t rle_pos = s->rle_pos;
  uint8_t* dst     = s->rle_out;

  while (rle_pos < numvals) {
    uint32_t pos    = rle_pos + t;
    uint32_t v      = (pos < numvals) ? s->vals[rolling_idx(pos)] : 0;
    uint32_t n      = min(numvals - rle_pos, 128);
    uint32_t nbytes = (n + ((flush) ? 7 : 0)) >> 3;
    if (!nbytes) { break; }
    v |= shuffle_xor(v, 1) << 1;
    v |= shuffle_xor(v, 2) << 2;
    v |= shuffle_xor(v, 4) << 4;
    if (t < n && !(t & 7)) { dst[t >> 3] = v; }
    rle_pos = min(rle_pos + nbytes * 8, numvals);
    dst += nbytes;
  }
  __syncthreads();
  if (!t) {
    s->rle_pos     = rle_pos;
    s->rle_numvals = numvals;
    s->rle_out     = dst;
  }
}

/**
 * @brief Determines the difference between the Proleptic Gregorian Calendar epoch (1970-01-01
 * 00:00:00 UTC) and the Julian date epoch (-4713-11-24 12:00:00 UTC).
 *
 * @return The difference between two epochs in `cuda::std::chrono::duration` format with a period
 * of hours.
 */
constexpr auto julian_calendar_epoch_diff()
{
  using namespace cuda::std::chrono;
  using namespace cuda::std::chrono_literals;
  return sys_days{January / 1 / 1970} - (sys_days{November / 24 / -4713} + 12h);
}

/**
 * @brief Converts number `v` of periods of type `PeriodT` into a pair with nanoseconds since
 * midnight and number of Julian days. Does not deal with time zones. Used by INT96 code.
 *
 * @tparam PeriodT a ratio representing the tick period in duration
 * @param v count of ticks since epoch
 * @return A pair of (nanoseconds, days) where nanoseconds is the number of nanoseconds
 * elapsed in the day and days is the number of days from Julian epoch.
 */
template <typename PeriodT>
__device__ auto julian_days_with_time(int64_t v)
{
  using namespace cuda::std::chrono;
  auto const dur_total             = duration<int64_t, PeriodT>{v};
  auto const dur_days              = floor<days>(dur_total);
  auto const dur_time_of_day       = dur_total - dur_days;
  auto const dur_time_of_day_nanos = duration_cast<nanoseconds>(dur_time_of_day);
  auto const julian_days           = dur_days + ceil<days>(julian_calendar_epoch_diff());
  return std::make_pair(dur_time_of_day_nanos, julian_days);
}

// this has been split out into its own kernel because of the amount of shared memory required
// for the state buffer. encode kernels that don't use the RLE buffer can get started while
// the level data is encoded.
// blockDim(128, 1, 1)
template <int block_size>
CUDF_KERNEL void __launch_bounds__(block_size, 8)
  gpuEncodePageLevels(device_span<EncPage> pages,
                      bool write_v2_headers,
                      encode_kernel_mask kernel_mask)
{
  __shared__ __align__(8) rle_page_enc_state_s state_g;

  auto* const s    = &state_g;
  uint32_t const t = threadIdx.x;

  if (t == 0) {
    state_g = rle_page_enc_state_s{};
    s->page = pages[blockIdx.x];
    s->ck   = *s->page.chunk;
    s->col  = *s->ck.col_desc;
    s->cur  = s->page.page_data + s->page.max_hdr_size;
    // init V2 info
    s->page.def_lvl_bytes = 0;
    s->page.rep_lvl_bytes = 0;
    s->page.num_nulls     = 0;
    s->rle_len_pos        = nullptr;
  }
  __syncthreads();

  if (BitAnd(s->page.kernel_mask, kernel_mask) == 0) { return; }

  auto const is_v2 = s->page.page_type == PageType::DATA_PAGE_V2;

  // Encode Repetition and Definition levels
  if (s->page.page_type != PageType::DICTIONARY_PAGE &&
      (s->col.num_def_level_bits()) != 0 &&  // This means max definition level is not 0 (nullable)
      (s->col.num_rep_level_bits()) == 0     // This means there are no repetition levels (non-list)
  ) {
    // Calculate definition levels from validity
    uint32_t def_lvl_bits = s->col.num_def_level_bits();
    if (def_lvl_bits != 0) {
      if (!t) {
        s->rle_run     = 0;
        s->rle_pos     = 0;
        s->rle_numvals = 0;
        s->rle_out     = s->cur;
        if (not is_v2) {
          s->rle_out += 4;  // save space for length
        }
      }
      __syncthreads();
      while (s->rle_numvals < s->page.num_rows) {
        uint32_t rle_numvals = s->rle_numvals;
        uint32_t nrows       = min(s->page.num_rows - rle_numvals, 128);
        auto row             = s->page.start_row + rle_numvals + t;
        // Definition level encodes validity. Checks the valid map and if it is valid, then sets the
        // def_lvl accordingly and sets it in s->vals which is then given to RleEncode to encode
        uint32_t def_lvl = [&]() {
          bool within_bounds = rle_numvals + t < s->page.num_rows && row < s->col.num_rows;
          if (not within_bounds) { return 0u; }
          uint32_t def       = 0;
          size_type l        = 0;
          bool is_col_struct = false;
          auto col           = *s->col.parent_column;
          do {
            // If col not nullable then it does not contribute to def levels
            if (s->col.nullability[l]) {
              if (col.is_valid(row)) {
                ++def;
              } else {
                // We have found the shallowest level at which this row is null
                break;
              }
            }
            is_col_struct = (col.type().id() == type_id::STRUCT);
            if (is_col_struct) {
              row += col.offset();
              col = col.child(0);
              ++l;
            }
          } while (is_col_struct);
          return def;
        }();
        s->vals[rolling_idx(rle_numvals + t)] = def_lvl;
        __syncthreads();
        // if max_def <= 1, then the histogram is trivial to calculate
        if (s->page.def_histogram != nullptr and s->col.max_def_level > 1) {
          // Only calculate up to max_def_level...the last entry is num_valid and will be filled
          // in later.
          generate_def_level_histogram<block_size>(
            s->page.def_histogram, s, nrows, rle_numvals, s->col.max_def_level);
        }
        rle_numvals += nrows;
        RleEncode(s, rle_numvals, def_lvl_bits, (rle_numvals == s->page.num_rows), t);
        __syncthreads();
      }
      if (t < 32) {
        uint8_t* const cur     = s->cur;
        uint8_t* const rle_out = s->rle_out;
        // V2 does not write the RLE length field
        uint32_t const rle_bytes =
          static_cast<uint32_t>(rle_out - cur) - (is_v2 ? 0 : RLE_LENGTH_FIELD_LEN);
        if (not is_v2 && t < RLE_LENGTH_FIELD_LEN) { cur[t] = rle_bytes >> (t * 8); }
        __syncwarp();
        if (t == 0) {
          s->cur                = rle_out;
          s->page.def_lvl_bytes = rle_bytes;
        }
      }
    }
  } else if (s->page.page_type != PageType::DICTIONARY_PAGE &&
             s->col.num_rep_level_bits() != 0  // This means there ARE repetition levels (has list)
  ) {
    auto encode_levels = [&](uint8_t const* lvl_val_data, uint32_t nbits, uint32_t& lvl_bytes) {
      // For list types, the repetition and definition levels are pre-calculated. We just need to
      // encode and write them now.
      if (!t) {
        s->rle_run     = 0;
        s->rle_pos     = 0;
        s->rle_numvals = 0;
        s->rle_out     = s->cur;
        if (not is_v2) {
          s->rle_out += 4;  // save space for length
        }
      }
      __syncthreads();
      size_type page_first_val_idx = s->col.level_offsets[s->page.start_row];
      size_type col_last_val_idx   = s->col.level_offsets[s->col.num_rows];
      while (s->rle_numvals < s->page.num_values) {
        uint32_t rle_numvals = s->rle_numvals;
        uint32_t nvals       = min(s->page.num_values - rle_numvals, 128);
        uint32_t idx         = page_first_val_idx + rle_numvals + t;
        uint32_t lvl_val =
          (rle_numvals + t < s->page.num_values && idx < col_last_val_idx) ? lvl_val_data[idx] : 0;
        s->vals[rolling_idx(rle_numvals + t)] = lvl_val;
        __syncthreads();
        rle_numvals += nvals;
        RleEncode(s, rle_numvals, nbits, (rle_numvals == s->page.num_values), t);
        __syncthreads();
      }
      if (t < 32) {
        uint8_t* const cur     = s->cur;
        uint8_t* const rle_out = s->rle_out;
        // V2 does not write the RLE length field
        uint32_t const rle_bytes =
          static_cast<uint32_t>(rle_out - cur) - (is_v2 ? 0 : RLE_LENGTH_FIELD_LEN);
        if (not is_v2 && t < RLE_LENGTH_FIELD_LEN) { cur[t] = rle_bytes >> (t * 8); }
        __syncwarp();
        if (t == 0) {
          s->cur    = rle_out;
          lvl_bytes = rle_bytes;
        }
      }
    };
    encode_levels(s->col.rep_values, s->col.num_rep_level_bits(), s->page.rep_lvl_bytes);
    __syncthreads();
    encode_levels(s->col.def_values, s->col.num_def_level_bits(), s->page.def_lvl_bytes);
  }

  if (t == 0) { pages[blockIdx.x] = s->page; }
}

template <int block_size, typename state_buf>
__device__ void finish_page_encode(state_buf* s,
                                   uint8_t const* end_ptr,
                                   device_span<EncPage> pages,
                                   device_span<device_span<uint8_t const>> comp_in,
                                   device_span<device_span<uint8_t>> comp_out,
                                   device_span<compression_result> comp_results,
                                   bool write_v2_headers)
{
  auto const t = threadIdx.x;

  // returns sum of histogram values from [1..max_level)
  auto histogram_sum = [](uint32_t* const hist, int max_level) {
    auto const hist_start = hist + 1;
    auto const hist_end   = hist + max_level;
    return thrust::reduce(thrust::seq, hist_start, hist_end, 0U);
  };

  // this will be true if max_rep > 0 (i.e. there are lists)
  if (s->page.rep_histogram != nullptr) {
    // for repetition we get hist[0] from num_rows, and can derive hist[max_rep_level]
    if (s->col.max_rep_level > 1) {
      generate_page_histogram<block_size>(
        s->page.rep_histogram, s, s->col.rep_values, s->col.max_rep_level);
    }

    if (t == 0) {
      // rep_hist[0] is num_rows, we have rep_hist[1..max_rep_level) calculated, so
      // rep_hist[max_rep_level] is num_values minus the sum of the preceding values.
      s->page.rep_histogram[0] = s->page.num_rows;
      s->page.rep_histogram[s->col.max_rep_level] =
        s->page.num_values - s->page.num_rows -
        histogram_sum(s->page.rep_histogram, s->col.max_rep_level);
    }
    __syncthreads();

    if (s->page.def_histogram != nullptr) {
      // For definition, we know `hist[max_def_level] = num_valid`. If the leaf level is
      // nullable, then `hist[max_def_level - 1] = num_leaf_values - num_valid`. Finally,
      // hist[0] can be derived as `num_values - sum(hist[1]..hist[max_def_level])`.
      bool const is_leaf_nullable = s->col.leaf_column->nullable();
      auto const last_lvl = is_leaf_nullable ? s->col.max_def_level - 1 : s->col.max_def_level;
      if (last_lvl > 1) {
        generate_page_histogram<block_size>(s->page.def_histogram, s, s->col.def_values, last_lvl);
      }

      if (t == 0) {
        s->page.def_histogram[s->col.max_def_level] = s->page.num_valid;
        if (is_leaf_nullable) {
          s->page.def_histogram[last_lvl] = s->page.num_leaf_values - s->page.num_valid;
        }
        s->page.def_histogram[0] = s->page.num_values - s->page.num_leaf_values -
                                   histogram_sum(s->page.def_histogram, last_lvl);
      }
    }
  } else if (s->page.def_histogram != nullptr) {
    // finish off what was started in generate_def_level_histogram
    if (t == 0) {
      // `hist[max_def_level] = num_valid`, and the values for hist[1..max_def_level) are known
      s->page.def_histogram[s->col.max_def_level] = s->page.num_valid;
      s->page.def_histogram[0]                    = s->page.num_values - s->page.num_valid -
                                 histogram_sum(s->page.def_histogram, s->col.max_def_level);
    }
  }

  if (t == 0) {
    // only need num_nulls for v2 data page headers
    if (write_v2_headers) { s->page.num_nulls = s->page.num_values - s->page.num_valid; }
    uint8_t const* const base   = s->page.page_data + s->page.max_hdr_size;
    auto const actual_data_size = static_cast<uint32_t>(end_ptr - base);
    if (actual_data_size > s->page.max_data_size) {
      // FIXME(ets): this needs to do error propagation back to the host
      CUDF_UNREACHABLE("detected possible page data corruption");
    }
    if (s->page.is_v2()) {
      auto const d_base = base + s->page.max_lvl_size;
      s->page.data_size = static_cast<uint32_t>(end_ptr - d_base) + s->page.level_bytes();
    } else {
      s->page.data_size = actual_data_size;
    }
    if (not comp_in.empty()) {
      auto const c_base            = base + s->page.max_lvl_size;
      auto const bytes_to_compress = static_cast<uint32_t>(end_ptr - c_base);
      comp_in[blockIdx.x]          = {c_base, bytes_to_compress};
      comp_out[blockIdx.x] = {s->page.compressed_data + s->page.max_hdr_size + s->page.max_lvl_size,
                              0};  // size is unused
    }
    pages[blockIdx.x] = s->page;
    if (not comp_results.empty()) {
      comp_results[blockIdx.x]   = {0, compression_status::FAILURE};
      pages[blockIdx.x].comp_res = &comp_results[blockIdx.x];
    }
  }

  // copy uncompressed bytes over
  if (s->page.is_v2() and not comp_in.empty()) {
    uint8_t* const src = s->page.page_data + s->page.max_hdr_size;
    uint8_t* const dst = s->page.compressed_data + s->page.max_hdr_size;
    for (int i = t; i < s->page.level_bytes(); i += block_size) {
      dst[i] = src[i];
    }
  }
}

// PLAIN page data encoder
// blockDim(128, 1, 1)
template <int block_size>
CUDF_KERNEL void __launch_bounds__(block_size, 8)
  gpuEncodePages(device_span<EncPage> pages,
                 device_span<device_span<uint8_t const>> comp_in,
                 device_span<device_span<uint8_t>> comp_out,
                 device_span<compression_result> comp_results,
                 bool write_v2_headers)
{
  __shared__ __align__(8) page_enc_state_s<0> state_g;
  using block_scan = hipcub::BlockScan<uint32_t, block_size>;
  __shared__ typename block_scan::TempStorage scan_storage;

  auto* const s = &state_g;
  uint32_t t    = threadIdx.x;

  if (t == 0) {
    state_g        = page_enc_state_s<0>{};
    s->page        = pages[blockIdx.x];
    s->ck          = *s->page.chunk;
    s->col         = *s->ck.col_desc;
    s->rle_len_pos = nullptr;
    // get s->cur back to where it was at the end of encoding the rep and def level data
    set_page_data_start(s);
  }
  __syncthreads();

  if (BitAnd(s->page.kernel_mask, encode_kernel_mask::PLAIN) == 0) { return; }

  // Encode data values
  __syncthreads();
  auto const physical_type = s->col.physical_type;
  auto const type_id       = s->col.leaf_column->type().id();
  auto const dtype_len_out = physical_type_len(physical_type, type_id);
  auto const dtype_len_in  = [&]() -> uint32_t {
    if (physical_type == INT32) { return int32_logical_len(type_id); }
    if (physical_type == INT96) { return sizeof(int64_t); }
    return dtype_len_out;
  }();

  if (t == 0) {
    uint8_t* dst   = s->cur;
    s->rle_run     = 0;
    s->rle_pos     = 0;
    s->rle_numvals = 0;
    s->rle_out     = dst;
    s->page.encoding =
      determine_encoding(s->page.page_type, physical_type, s->ck.use_dictionary, write_v2_headers);
    s->page_start_val  = row_to_value_idx(s->page.start_row, s->col);
    s->chunk_start_val = row_to_value_idx(s->ck.start_row, s->col);
  }
  __syncthreads();

  for (uint32_t cur_val_idx = 0; cur_val_idx < s->page.num_leaf_values;) {
    uint32_t nvals = min(s->page.num_leaf_values - cur_val_idx, block_size);
    uint32_t len, pos;

    auto [is_valid, val_idx] = [&]() {
      uint32_t val_idx;
      uint32_t is_valid;

      size_type const val_idx_in_block = cur_val_idx + t;
      if (s->page.page_type == PageType::DICTIONARY_PAGE) {
        val_idx  = val_idx_in_block;
        is_valid = (val_idx < s->page.num_leaf_values);
        if (is_valid) { val_idx = s->ck.dict_data[val_idx]; }
      } else {
        size_type const val_idx_in_leaf_col = s->page_start_val + val_idx_in_block;

        is_valid = (val_idx_in_leaf_col < s->col.leaf_column->size() &&
                    val_idx_in_block < s->page.num_leaf_values)
                     ? s->col.leaf_column->is_valid(val_idx_in_leaf_col)
                     : 0;
        val_idx  = val_idx_in_leaf_col;
      }
      return std::make_tuple(is_valid, val_idx);
    }();

    cur_val_idx += nvals;

    // Non-dictionary encoding
    uint8_t* dst = s->cur;

    if (is_valid) {
      len = dtype_len_out;
      if (physical_type == BYTE_ARRAY) {
        if (type_id == type_id::STRING) {
          len += s->col.leaf_column->element<string_view>(val_idx).size_bytes();
        } else if (s->col.output_as_byte_array && type_id == type_id::LIST) {
          len +=
            get_element<statistics::byte_array_view>(*s->col.leaf_column, val_idx).size_bytes();
        }
      }
    } else {
      len = 0;
    }
    uint32_t total_len = 0;
    block_scan(scan_storage).ExclusiveSum(len, pos, total_len);
    __syncthreads();
    if (t == 0) { s->cur = dst + total_len; }
    if (is_valid) {
      switch (physical_type) {
        case INT32: [[fallthrough]];
        case FLOAT: {
          auto const v = [dtype_len = dtype_len_in,
                          idx       = val_idx,
                          col       = s->col.leaf_column,
                          scale     = s->col.ts_scale == 0 ? 1 : s->col.ts_scale]() -> int32_t {
            switch (dtype_len) {
              case 8: return col->element<int64_t>(idx) * scale;
              case 4: return col->element<int32_t>(idx) * scale;
              case 2: return col->element<int16_t>(idx) * scale;
              default: return col->element<int8_t>(idx) * scale;
            }
          }();

          dst[pos + 0] = v;
          dst[pos + 1] = v >> 8;
          dst[pos + 2] = v >> 16;
          dst[pos + 3] = v >> 24;
        } break;
        case INT64: {
          int64_t v        = s->col.leaf_column->element<int64_t>(val_idx);
          int32_t ts_scale = s->col.ts_scale;
          if (ts_scale != 0) {
            if (ts_scale < 0) {
              v /= -ts_scale;
            } else {
              v *= ts_scale;
            }
          }
          dst[pos + 0] = v;
          dst[pos + 1] = v >> 8;
          dst[pos + 2] = v >> 16;
          dst[pos + 3] = v >> 24;
          dst[pos + 4] = v >> 32;
          dst[pos + 5] = v >> 40;
          dst[pos + 6] = v >> 48;
          dst[pos + 7] = v >> 56;
        } break;
        case INT96: {
          int64_t v        = s->col.leaf_column->element<int64_t>(val_idx);
          int32_t ts_scale = s->col.ts_scale;
          if (ts_scale != 0) {
            if (ts_scale < 0) {
              v /= -ts_scale;
            } else {
              v *= ts_scale;
            }
          }

          auto const [last_day_nanos, julian_days] = [&] {
            using namespace cuda::std::chrono;
            switch (s->col.leaf_column->type().id()) {
              case type_id::TIMESTAMP_SECONDS:
              case type_id::TIMESTAMP_MILLISECONDS: {
                return julian_days_with_time<cuda::std::milli>(v);
              } break;
              case type_id::TIMESTAMP_MICROSECONDS:
              case type_id::TIMESTAMP_NANOSECONDS: {
                return julian_days_with_time<cuda::std::micro>(v);
              } break;
            }
            return julian_days_with_time<cuda::std::nano>(0);
          }();

          // the 12 bytes of fixed length data.
          v             = last_day_nanos.count();
          dst[pos + 0]  = v;
          dst[pos + 1]  = v >> 8;
          dst[pos + 2]  = v >> 16;
          dst[pos + 3]  = v >> 24;
          dst[pos + 4]  = v >> 32;
          dst[pos + 5]  = v >> 40;
          dst[pos + 6]  = v >> 48;
          dst[pos + 7]  = v >> 56;
          uint32_t w    = julian_days.count();
          dst[pos + 8]  = w;
          dst[pos + 9]  = w >> 8;
          dst[pos + 10] = w >> 16;
          dst[pos + 11] = w >> 24;
        } break;

        case DOUBLE: {
          auto v = s->col.leaf_column->element<double>(val_idx);
          memcpy(dst + pos, &v, 8);
        } break;
        case BYTE_ARRAY: {
          auto const bytes = [](cudf::type_id const type_id,
                                column_device_view const* leaf_column,
                                uint32_t const val_idx) -> void const* {
            switch (type_id) {
              case type_id::STRING:
                return reinterpret_cast<void const*>(
                  leaf_column->element<string_view>(val_idx).data());
              case type_id::LIST:
                return reinterpret_cast<void const*>(
                  get_element<statistics::byte_array_view>(*(leaf_column), val_idx).data());
              default: CUDF_UNREACHABLE("invalid type id for byte array writing!");
            }
          }(type_id, s->col.leaf_column, val_idx);
          uint32_t v   = len - 4;  // string length
          dst[pos + 0] = v;
          dst[pos + 1] = v >> 8;
          dst[pos + 2] = v >> 16;
          dst[pos + 3] = v >> 24;
          if (v != 0) memcpy(dst + pos + 4, bytes, v);
        } break;
        case FIXED_LEN_BYTE_ARRAY: {
          if (type_id == type_id::DECIMAL128) {
            // When using FIXED_LEN_BYTE_ARRAY for decimals, the rep is encoded in big-endian
            auto const v = s->col.leaf_column->element<numeric::decimal128>(val_idx).value();
            auto const v_char_ptr = reinterpret_cast<char const*>(&v);
            thrust::copy(thrust::seq,
                         thrust::make_reverse_iterator(v_char_ptr + sizeof(v)),
                         thrust::make_reverse_iterator(v_char_ptr),
                         dst + pos);
          }
        } break;
      }
    }
    __syncthreads();
  }

  finish_page_encode<block_size>(
    s, s->cur, pages, comp_in, comp_out, comp_results, write_v2_headers);
}

// DICTIONARY page data encoder
// blockDim(128, 1, 1)
template <int block_size>
CUDF_KERNEL void __launch_bounds__(block_size, 8)
  gpuEncodeDictPages(device_span<EncPage> pages,
                     device_span<device_span<uint8_t const>> comp_in,
                     device_span<device_span<uint8_t>> comp_out,
                     device_span<compression_result> comp_results,
                     bool write_v2_headers)
{
  __shared__ __align__(8) rle_page_enc_state_s state_g;
  using block_scan = hipcub::BlockScan<uint32_t, block_size>;
  __shared__ typename block_scan::TempStorage scan_storage;

  auto* const s = &state_g;
  uint32_t t    = threadIdx.x;

  if (t == 0) {
    state_g        = rle_page_enc_state_s{};
    s->page        = pages[blockIdx.x];
    s->ck          = *s->page.chunk;
    s->col         = *s->ck.col_desc;
    s->rle_len_pos = nullptr;
    // get s->cur back to where it was at the end of encoding the rep and def level data
    set_page_data_start(s);
  }
  __syncthreads();

  if (BitAnd(s->page.kernel_mask, encode_kernel_mask::DICTIONARY) == 0) { return; }

  // Encode data values
  auto const physical_type = s->col.physical_type;
  auto const type_id       = s->col.leaf_column->type().id();
  auto const dtype_len_out = physical_type_len(physical_type, type_id);
  auto const dtype_len_in  = [&]() -> uint32_t {
    if (physical_type == INT32) { return int32_logical_len(type_id); }
    if (physical_type == INT96) { return sizeof(int64_t); }
    return dtype_len_out;
  }();

  // TODO assert dict_bits >= 0
  auto const dict_bits = (physical_type == BOOLEAN) ? 1
                         : (s->ck.use_dictionary and s->page.page_type != PageType::DICTIONARY_PAGE)
                           ? s->ck.dict_rle_bits
                           : -1;
  if (t == 0) {
    uint8_t* dst   = s->cur;
    s->rle_run     = 0;
    s->rle_pos     = 0;
    s->rle_numvals = 0;
    s->rle_out     = dst;
    s->page.encoding =
      determine_encoding(s->page.page_type, physical_type, s->ck.use_dictionary, write_v2_headers);
    if (dict_bits >= 0 && physical_type != BOOLEAN) {
      dst[0]     = dict_bits;
      s->rle_out = dst + 1;
    } else if (write_v2_headers && physical_type == BOOLEAN) {
      // save space for RLE length. we don't know the total length yet.
      s->rle_out     = dst + RLE_LENGTH_FIELD_LEN;
      s->rle_len_pos = dst;
    }
    s->page_start_val  = row_to_value_idx(s->page.start_row, s->col);
    s->chunk_start_val = row_to_value_idx(s->ck.start_row, s->col);
  }
  __syncthreads();

  for (uint32_t cur_val_idx = 0; cur_val_idx < s->page.num_leaf_values;) {
    uint32_t nvals = min(s->page.num_leaf_values - cur_val_idx, block_size);

    auto [is_valid, val_idx] = [&]() {
      size_type const val_idx_in_block    = cur_val_idx + t;
      size_type const val_idx_in_leaf_col = s->page_start_val + val_idx_in_block;

      uint32_t const is_valid = (val_idx_in_leaf_col < s->col.leaf_column->size() &&
                                 val_idx_in_block < s->page.num_leaf_values)
                                  ? s->col.leaf_column->is_valid(val_idx_in_leaf_col)
                                  : 0;
      // need to test for use_dictionary because it might be boolean
      uint32_t const val_idx =
        (s->ck.use_dictionary) ? val_idx_in_leaf_col - s->chunk_start_val : val_idx_in_leaf_col;
      return std::make_tuple(is_valid, val_idx);
    }();

    cur_val_idx += nvals;

    // Dictionary encoding
    if (dict_bits > 0) {
      uint32_t rle_numvals;
      uint32_t rle_numvals_in_block;
      uint32_t pos;
      block_scan(scan_storage).ExclusiveSum(is_valid, pos, rle_numvals_in_block);
      rle_numvals = s->rle_numvals;
      if (is_valid) {
        uint32_t v;
        if (physical_type == BOOLEAN) {
          v = s->col.leaf_column->element<uint8_t>(val_idx);
        } else {
          v = s->ck.dict_index[val_idx];
        }
        s->vals[rolling_idx(rle_numvals + pos)] = v;
      }
      rle_numvals += rle_numvals_in_block;
      __syncthreads();
      if ((!write_v2_headers) && (physical_type == BOOLEAN)) {
        PlainBoolEncode(s, rle_numvals, (cur_val_idx == s->page.num_leaf_values), t);
      } else {
        RleEncode(s, rle_numvals, dict_bits, (cur_val_idx == s->page.num_leaf_values), t);
      }
      __syncthreads();
    }
    if (t == 0) { s->cur = s->rle_out; }
    __syncthreads();
  }

  // save RLE length if necessary
  if (s->rle_len_pos != nullptr && t < 32) {
    // size doesn't include the 4 bytes for the length
    auto const rle_size = static_cast<uint32_t>(s->cur - s->rle_len_pos) - RLE_LENGTH_FIELD_LEN;
    if (t < RLE_LENGTH_FIELD_LEN) { s->rle_len_pos[t] = rle_size >> (t * 8); }
    __syncwarp();
  }

  finish_page_encode<block_size>(
    s, s->cur, pages, comp_in, comp_out, comp_results, write_v2_headers);
}

// DELTA_BINARY_PACKED page data encoder
// blockDim(128, 1, 1)
template <int block_size>
CUDF_KERNEL void __launch_bounds__(block_size, 8)
  gpuEncodeDeltaBinaryPages(device_span<EncPage> pages,
                            device_span<device_span<uint8_t const>> comp_in,
                            device_span<device_span<uint8_t>> comp_out,
                            device_span<compression_result> comp_results)
{
  // block of shared memory for value storage and bit packing
  __shared__ uleb128_t delta_shared[delta::buffer_size + delta::block_size];
  __shared__ __align__(8) page_enc_state_s<0> state_g;
  __shared__ union {
    typename delta_binary_packer<uleb128_t>::index_scan::TempStorage delta_index_tmp;
    typename delta_binary_packer<uleb128_t>::block_reduce::TempStorage delta_reduce_tmp;
    typename delta_binary_packer<uleb128_t>::warp_reduce::TempStorage
      delta_warp_red_tmp[delta::num_mini_blocks];
  } temp_storage;

  auto* const s = &state_g;
  uint32_t t    = threadIdx.x;

  if (t == 0) {
    state_g        = page_enc_state_s<0>{};
    s->page        = pages[blockIdx.x];
    s->ck          = *s->page.chunk;
    s->col         = *s->ck.col_desc;
    s->rle_len_pos = nullptr;
    // get s->cur back to where it was at the end of encoding the rep and def level data
    set_page_data_start(s);
  }
  __syncthreads();

  if (BitAnd(s->page.kernel_mask, encode_kernel_mask::DELTA_BINARY) == 0) { return; }

  // Encode data values
  auto const physical_type = s->col.physical_type;
  auto const type_id       = s->col.leaf_column->type().id();
  auto const dtype_len_out = physical_type_len(physical_type, type_id);
  auto const dtype_len_in  = [&]() -> uint32_t {
    if (physical_type == INT32) { return int32_logical_len(type_id); }
    if (physical_type == INT96) { return sizeof(int64_t); }
    return dtype_len_out;
  }();

  if (t == 0) {
    uint8_t* dst       = s->cur;
    s->rle_run         = 0;
    s->rle_pos         = 0;
    s->rle_numvals     = 0;
    s->rle_out         = dst;
    s->page.encoding   = Encoding::DELTA_BINARY_PACKED;
    s->page_start_val  = row_to_value_idx(s->page.start_row, s->col);
    s->chunk_start_val = row_to_value_idx(s->ck.start_row, s->col);
  }
  __syncthreads();

  uint8_t const* delta_ptr = nullptr;  // this will be the end of delta block pointer

  if (physical_type == INT32) {
    switch (dtype_len_in) {
      case 8: {
        // only DURATIONS map to 8 bytes, so safe to just use signed here?
        delta_ptr = delta_encode<INT32, int64_t>(s, delta_shared, &temp_storage);
        break;
      }
      case 4: {
        if (type_id == type_id::UINT32) {
          delta_ptr = delta_encode<INT32, uint32_t>(s, delta_shared, &temp_storage);
        } else {
          delta_ptr = delta_encode<INT32, int32_t>(s, delta_shared, &temp_storage);
        }
        break;
      }
      case 2: {
        if (type_id == type_id::UINT16) {
          delta_ptr = delta_encode<INT32, uint16_t>(s, delta_shared, &temp_storage);
        } else {
          delta_ptr = delta_encode<INT32, int16_t>(s, delta_shared, &temp_storage);
        }
        break;
      }
      case 1: {
        if (type_id == type_id::UINT8) {
          delta_ptr = delta_encode<INT32, uint8_t>(s, delta_shared, &temp_storage);
        } else {
          delta_ptr = delta_encode<INT32, int8_t>(s, delta_shared, &temp_storage);
        }
        break;
      }
      default: CUDF_UNREACHABLE("invalid dtype_len_in when encoding DELTA_BINARY_PACKED");
    }
  } else {
    if (type_id == type_id::UINT64) {
      delta_ptr = delta_encode<INT64, uint64_t>(s, delta_shared, &temp_storage);
    } else {
      delta_ptr = delta_encode<INT64, int64_t>(s, delta_shared, &temp_storage);
    }
  }

  finish_page_encode<block_size>(s, delta_ptr, pages, comp_in, comp_out, comp_results, true);
}

// DELTA_LENGTH_BYTE_ARRAY page data encoder
// blockDim(128, 1, 1)
template <int block_size>
CUDF_KERNEL void __launch_bounds__(block_size, 8)
  gpuEncodeDeltaLengthByteArrayPages(device_span<EncPage> pages,
                                     device_span<device_span<uint8_t const>> comp_in,
                                     device_span<device_span<uint8_t>> comp_out,
                                     device_span<compression_result> comp_results)
{
  // block of shared memory for value storage and bit packing
  __shared__ uleb128_t delta_shared[delta::buffer_size + delta::block_size];
  __shared__ __align__(8) page_enc_state_s<0> state_g;
  __shared__ delta_binary_packer<int32_t> packer;
  __shared__ uint8_t const* first_string;
  __shared__ size_type string_data_len;
  using block_reduce = hipcub::BlockReduce<uint32_t, block_size>;
  __shared__ union {
    typename block_reduce::TempStorage reduce_storage;
    typename delta_binary_packer<uleb128_t>::index_scan::TempStorage delta_index_tmp;
    typename delta_binary_packer<uleb128_t>::block_reduce::TempStorage delta_reduce_tmp;
    typename delta_binary_packer<uleb128_t>::warp_reduce::TempStorage
      delta_warp_red_tmp[delta::num_mini_blocks];
  } temp_storage;

  auto* const s = &state_g;
  uint32_t t    = threadIdx.x;

  if (t == 0) {
    state_g        = page_enc_state_s<0>{};
    s->page        = pages[blockIdx.x];
    s->ck          = *s->page.chunk;
    s->col         = *s->ck.col_desc;
    s->rle_len_pos = nullptr;
    // get s->cur back to where it was at the end of encoding the rep and def level data
    set_page_data_start(s);
  }
  __syncthreads();

  if (BitAnd(s->page.kernel_mask, encode_kernel_mask::DELTA_LENGTH_BA) == 0) { return; }

  // Encode data values
  if (t == 0) {
    uint8_t* dst       = s->cur;
    s->rle_run         = 0;
    s->rle_pos         = 0;
    s->rle_numvals     = 0;
    s->rle_out         = dst;
    s->page.encoding   = Encoding::DELTA_LENGTH_BYTE_ARRAY;
    s->page_start_val  = row_to_value_idx(s->page.start_row, s->col);
    s->chunk_start_val = row_to_value_idx(s->ck.start_row, s->col);
  }
  __syncthreads();

  auto const type_id = s->col.leaf_column->type().id();

  // encode the lengths as DELTA_BINARY_PACKED
  if (t == 0) {
    first_string = nullptr;
    packer.init(s->cur, s->page.num_valid, reinterpret_cast<int32_t*>(delta_shared), &temp_storage);

    // if there are valid values, find a pointer to the first valid string
    if (s->page.num_valid != 0) {
      for (uint32_t idx = 0; idx < s->page.num_leaf_values; idx++) {
        size_type const idx_in_col = s->page_start_val + idx;
        if (s->col.leaf_column->is_valid(idx_in_col)) {
          if (type_id == type_id::STRING) {
            first_string = reinterpret_cast<uint8_t const*>(
              s->col.leaf_column->element<string_view>(idx_in_col).data());
          } else if (s->col.output_as_byte_array && type_id == type_id::LIST) {
            first_string = reinterpret_cast<uint8_t const*>(
              get_element<statistics::byte_array_view>(*s->col.leaf_column, idx_in_col).data());
          }
          break;
        }
      }
    }
  }
  __syncthreads();

  uint32_t len = 0;
  for (uint32_t cur_val_idx = 0; cur_val_idx < s->page.num_leaf_values;) {
    uint32_t const nvals = min(s->page.num_leaf_values - cur_val_idx, delta::block_size);

    size_type const val_idx_in_block = cur_val_idx + t;
    size_type const val_idx          = s->page_start_val + val_idx_in_block;

    bool const is_valid =
      (val_idx < s->col.leaf_column->size() && val_idx_in_block < s->page.num_leaf_values)
        ? s->col.leaf_column->is_valid(val_idx)
        : false;

    cur_val_idx += nvals;

    int32_t v = 0;
    if (is_valid) {
      if (type_id == type_id::STRING) {
        v = s->col.leaf_column->element<string_view>(val_idx).size_bytes();
      } else if (s->col.output_as_byte_array && type_id == type_id::LIST) {
        auto const arr_size =
          get_element<statistics::byte_array_view>(*s->col.leaf_column, val_idx).size_bytes();
        // the lengths are assumed to be INT32, check for overflow
        if (arr_size > static_cast<size_t>(std::numeric_limits<int32_t>::max())) {
          CUDF_UNREACHABLE("byte array size exceeds 2GB");
        }
        v = static_cast<int32_t>(arr_size);
      }
      len += v;
    }

    packer.add_value(v, is_valid);
  }

  // string_len is only valid on thread 0
  auto const string_len = block_reduce(temp_storage.reduce_storage).Sum(len);
  if (t == 0) { string_data_len = string_len; }
  __syncthreads();

  // finish off the delta block and get the pointer to the end of the delta block
  auto const output_ptr = packer.flush();

  // now copy the char data
  memcpy_block<block_size, true>(output_ptr, first_string, string_data_len, t);

  finish_page_encode<block_size>(
    s, output_ptr + string_data_len, pages, comp_in, comp_out, comp_results, true);
}

struct byte_array {
  uint8_t const* data;
  size_type length;

  // calculate the amount of overlap with a preceding array
  __device__ size_type common_prefix_length(byte_array const& preceding) const
  {
    auto const max_pref_len = min(length, preceding.length);
    size_type idx           = 0;
    while (idx < max_pref_len and data[idx] == preceding.data[idx]) {
      idx++;
    }
    return idx;
  }
};

// DELTA_BYTE_ARRAY page data encoder
// blockDim(128, 1, 1)
template <int block_size>
CUDF_KERNEL void __launch_bounds__(block_size, 8)
  gpuEncodeDeltaByteArrayPages(device_span<EncPage> pages,
                               device_span<device_span<uint8_t const>> comp_in,
                               device_span<device_span<uint8_t>> comp_out,
                               device_span<compression_result> comp_results)
{
  using cudf::detail::warp_size;
  // block of shared memory for value storage and bit packing
  __shared__ uleb128_t delta_shared[delta::buffer_size + delta::block_size];
  __shared__ __align__(8) page_enc_state_s<0> state_g;
  __shared__ delta_binary_packer<int32_t> packer;
  __shared__ uint8_t* scratch_data;
  __shared__ size_t avg_suffix_len;
  using block_scan   = hipcub::BlockScan<size_type, block_size>;
  using block_reduce = hipcub::BlockReduce<size_t, block_size>;
  __shared__ union {
    typename block_scan::TempStorage scan_storage;
    typename block_reduce::TempStorage reduce_storage;
    typename delta_binary_packer<uleb128_t>::index_scan::TempStorage delta_index_tmp;
    typename delta_binary_packer<uleb128_t>::block_reduce::TempStorage delta_reduce_tmp;
    typename delta_binary_packer<uleb128_t>::warp_reduce::TempStorage
      delta_warp_red_tmp[delta::num_mini_blocks];
  } temp_storage;

  auto* const s = &state_g;
  uint32_t t    = threadIdx.x;

  if (t == 0) {
    state_g        = page_enc_state_s<0>{};
    s->page        = pages[blockIdx.x];
    s->ck          = *s->page.chunk;
    s->col         = *s->ck.col_desc;
    s->rle_len_pos = nullptr;
    // get s->cur back to where it was at the end of encoding the rep and def level data
    set_page_data_start(s);
  }
  __syncthreads();

  if (BitAnd(s->page.kernel_mask, encode_kernel_mask::DELTA_BYTE_ARRAY) == 0) { return; }

  // Encode data values
  if (t == 0) {
    uint8_t* dst       = s->cur;
    s->rle_run         = 0;
    s->rle_pos         = 0;
    s->rle_numvals     = 0;
    s->rle_out         = dst;
    s->page.encoding   = Encoding::DELTA_BYTE_ARRAY;
    s->page_start_val  = row_to_value_idx(s->page.start_row, s->col);
    s->chunk_start_val = row_to_value_idx(s->ck.start_row, s->col);

    // set pointer to beginning of scratch space (aligned to size_type boundary)
    auto scratch_start =
      reinterpret_cast<uintptr_t>(s->page.page_data + s->page.max_hdr_size + s->page.max_data_size);
    scratch_start = util::round_up_unsafe(scratch_start, sizeof(size_type));
    scratch_data  = reinterpret_cast<uint8_t*>(scratch_start);
  }
  __syncthreads();

  // create offsets map (if needed)
  // We only encode valid values, and we need to know adjacent valid strings. So first we'll
  // create a mapping of leaf indexes to valid indexes:
  //
  // validity array is_valid:
  //   1 1 0 1 0 1 1 0
  //
  // exclusive scan on is_valid yields mapping of leaf index -> valid index:
  //   0 1 2 2 3 3 4 5
  //
  // Last value should equal page.num_valid. Now we need to transform that into a reverse
  // lookup that maps valid index -> leaf index (of length num_valid):
  //   0 1 3 5 6
  //
  auto const has_leaf_nulls = s->page.num_valid != s->page.num_leaf_values;

  size_type* const offsets_map =
    has_leaf_nulls ? reinterpret_cast<size_type*>(scratch_data) : nullptr;

  if (offsets_map != nullptr) {
    size_type* const forward_map = offsets_map + s->page.num_valid;

    // create the validity array
    for (int idx = t; idx < s->page.num_leaf_values; idx += block_size) {
      size_type const idx_in_col = s->page_start_val + idx;
      bool const is_valid =
        idx_in_col < s->col.leaf_column->size() and s->col.leaf_column->is_valid(idx_in_col);
      forward_map[idx] = is_valid ? 1 : 0;
    }
    __syncthreads();

    // exclusive scan to get leaf_idx -> valid_idx
    block_excl_sum<block_size>(forward_map, s->page.num_leaf_values, 0);

    // now reverse map to get valid_idx -> leaf_idx mapping
    for (int idx = t; idx < s->page.num_leaf_values; idx += block_size) {
      size_type const idx_in_col = s->page_start_val + idx;
      bool const is_valid =
        idx_in_col < s->col.leaf_column->size() and s->col.leaf_column->is_valid(idx_in_col);
      if (is_valid) { offsets_map[forward_map[idx]] = idx; }
    }
    __syncthreads();
  }

  size_type* const prefix_lengths =
    has_leaf_nulls ? offsets_map + s->page.num_valid : reinterpret_cast<size_type*>(scratch_data);

  auto const type_id = s->col.leaf_column->type().id();

  auto const byte_array_at = [type_id, s](size_type idx) -> byte_array {
    if (type_id == type_id::STRING) {
      auto const str = s->col.leaf_column->element<string_view>(idx);
      return {reinterpret_cast<uint8_t const*>(str.data()), str.size_bytes()};
    } else if (s->col.output_as_byte_array && type_id == type_id::LIST) {
      auto const str = get_element<statistics::byte_array_view>(*s->col.leaf_column, idx);
      return {reinterpret_cast<uint8_t const*>(str.data()),
              static_cast<size_type>(str.size_bytes())};
    }
    return {nullptr, 0};
  };

  // Calculate prefix lengths. The first prefix length is always 0. loop over num_valid since we
  // only encode valid values.
  // Note: calculating this on a string-per-thread basis seems bad for large strings with lots
  // of overlap. But in testing, it was found that the string copy at the end had a much larger
  // impact on performance, and doing this step on a string-per-warp basis was always slower.
  if (t == 0) { prefix_lengths[0] = 0; }
  for (int idx = t + 1; idx < s->page.num_valid; idx += block_size) {
    size_type const leaf_idx  = has_leaf_nulls ? offsets_map[idx] : idx;
    size_type const pleaf_idx = has_leaf_nulls ? offsets_map[idx - 1] : idx - 1;

    // get this string and the preceding string
    auto const current   = byte_array_at(leaf_idx + s->page_start_val);
    auto const preceding = byte_array_at(pleaf_idx + s->page_start_val);

    // calculate the amount of overlap
    prefix_lengths[idx] = current.common_prefix_length(preceding);
  }

  // encode prefix lengths
  if (t == 0) {
    packer.init(s->cur, s->page.num_valid, reinterpret_cast<int32_t*>(delta_shared), &temp_storage);
  }
  __syncthreads();

  // don't start at `t` because all threads must participate in each iteration
  for (int idx = 0; idx < s->page.num_valid; idx += block_size) {
    size_type const t_idx = idx + t;
    auto const in_range   = t_idx < s->page.num_valid;
    auto const val        = in_range ? prefix_lengths[t_idx] : 0;
    packer.add_value(val, in_range);
  }

  auto const suffix_ptr = packer.flush();
  __syncthreads();

  // encode suffix lengths
  if (t == 0) {
    packer.init(
      suffix_ptr, s->page.num_valid, reinterpret_cast<int32_t*>(delta_shared), &temp_storage);
  }
  __syncthreads();

  size_t non_zero     = 0;
  size_t suffix_bytes = 0;

  for (int idx = 0; idx < s->page.num_valid; idx += block_size) {
    size_type const t_idx = idx + t;
    auto const in_range   = t_idx < s->page.num_valid;
    int32_t val           = 0;
    if (in_range) {
      size_type const leaf_idx = has_leaf_nulls ? offsets_map[t_idx] : t_idx;
      auto const byte_arr      = byte_array_at(leaf_idx + s->page_start_val);
      val                      = byte_arr.length - prefix_lengths[t_idx];
      if (val > 0) {
        non_zero++;
        suffix_bytes += val;
      }
    }
    packer.add_value(val, in_range);
  }

  auto const strings_ptr = packer.flush();

  non_zero = block_reduce(temp_storage.reduce_storage).Sum(non_zero);
  __syncthreads();
  suffix_bytes = block_reduce(temp_storage.reduce_storage).Sum(suffix_bytes);
  if (t == 0) { avg_suffix_len = util::div_rounding_up_unsafe(suffix_bytes, non_zero); }
  __syncthreads();

  // Now copy the byte array data. For shorter suffixes (<= 64 bytes), it is faster to use
  // memcpy on a string-per-thread basis. For longer suffixes, it's better to use a parallel
  // approach. 64 was a good cutoff in testing.
  constexpr size_t suffix_cutoff = 64;

  size_t str_data_len = 0;
  if (avg_suffix_len <= suffix_cutoff) {
    for (int idx = 0; idx < s->page.num_valid; idx += block_size) {
      size_type const t_idx = idx + t;
      size_type s_len = 0, pref_len = 0, suff_len = 0;
      uint8_t const* s_ptr = nullptr;
      if (t_idx < s->page.num_valid) {
        size_type const leaf_idx = has_leaf_nulls ? offsets_map[t_idx] : t_idx;
        auto const byte_arr      = byte_array_at(leaf_idx + s->page_start_val);
        s_len                    = byte_arr.length;
        s_ptr                    = byte_arr.data;
        pref_len                 = prefix_lengths[t_idx];
        suff_len                 = byte_arr.length - pref_len;
      }

      // calculate offsets into output
      size_type s_off, total;
      block_scan(temp_storage.scan_storage)
        .ExclusiveScan(suff_len, s_off, str_data_len, hipcub::Sum(), total);

      if (t_idx < s->page.num_valid) {
        auto const dst = strings_ptr + s_off;
        memcpy(dst, s_ptr + pref_len, suff_len);
      }
      str_data_len += total;
      __syncthreads();
    }
  } else {
    int t0 = 0;  // thread 0 for each string
    for (int idx = 0; idx < s->page.num_valid; idx++) {
      // calculate ids for this string
      int const tid = (t - t0 + block_size) % block_size;

      // fetch string for this iter
      size_type const leaf_idx = has_leaf_nulls ? offsets_map[idx] : idx;
      auto const byte_arr      = byte_array_at(leaf_idx + s->page_start_val);
      size_type const pref_len = prefix_lengths[idx];
      size_type const suff_len = byte_arr.length - pref_len;

      // now copy the data
      auto const dst = strings_ptr + str_data_len;
      for (int src_idx = tid; src_idx < suff_len; src_idx += block_size) {
        dst[src_idx] = byte_arr.data[pref_len + src_idx];
      }

      str_data_len += suff_len;
      t0 = (t0 + suff_len) % block_size;
    }
  }

  finish_page_encode<block_size>(
    s, strings_ptr + str_data_len, pages, comp_in, comp_out, comp_results, true);
}

constexpr int decide_compression_warps_in_block = 4;
constexpr int decide_compression_block_size =
  decide_compression_warps_in_block * cudf::detail::warp_size;

// blockDim(decide_compression_block_size, 1, 1)
CUDF_KERNEL void __launch_bounds__(decide_compression_block_size)
  gpuDecideCompression(device_span<EncColumnChunk> chunks)
{
  __shared__ __align__(8) EncColumnChunk ck_g[decide_compression_warps_in_block];
  __shared__ __align__(4) unsigned int compression_error[decide_compression_warps_in_block];
  using warp_reduce = hipcub::WarpReduce<uint32_t>;
  __shared__ typename warp_reduce::TempStorage temp_storage[decide_compression_warps_in_block][2];

  auto const lane_id  = threadIdx.x % cudf::detail::warp_size;
  auto const warp_id  = threadIdx.x / cudf::detail::warp_size;
  auto const chunk_id = blockIdx.x * decide_compression_warps_in_block + warp_id;

  if (chunk_id >= chunks.size()) { return; }

  if (lane_id == 0) {
    ck_g[warp_id]              = chunks[chunk_id];
    compression_error[warp_id] = 0;
  }
  __syncwarp();

  uint32_t uncompressed_data_size = 0;
  uint32_t compressed_data_size   = 0;
  uint32_t encodings              = 0;
  auto const num_pages            = ck_g[warp_id].num_pages;
  for (auto page_id = lane_id; page_id < num_pages; page_id += cudf::detail::warp_size) {
    auto const& curr_page     = ck_g[warp_id].pages[page_id];
    auto const page_data_size = curr_page.data_size;
    uncompressed_data_size += page_data_size;
    if (auto comp_res = curr_page.comp_res; comp_res != nullptr) {
      auto const lvl_bytes = curr_page.is_v2() ? curr_page.level_bytes() : 0;
      compressed_data_size += comp_res->bytes_written + lvl_bytes;
      if (comp_res->status != compression_status::SUCCESS) {
        atomicOr(&compression_error[warp_id], 1);
      }
    }
    // collect encoding info for the chunk metadata
    encodings |= encoding_to_mask(curr_page.encoding);
  }
  uncompressed_data_size = warp_reduce(temp_storage[warp_id][0]).Sum(uncompressed_data_size);
  compressed_data_size   = warp_reduce(temp_storage[warp_id][1]).Sum(compressed_data_size);
  __syncwarp();
  encodings = warp_reduce(temp_storage[warp_id][0]).Reduce(encodings, BitwiseOr{});
  __syncwarp();

  if (lane_id == 0) {
    auto const write_compressed = compressed_data_size != 0 and compression_error[warp_id] == 0 and
                                  compressed_data_size < uncompressed_data_size;
    chunks[chunk_id].is_compressed = write_compressed;
    chunks[chunk_id].bfr_size      = uncompressed_data_size;
    chunks[chunk_id].compressed_size =
      write_compressed ? compressed_data_size : uncompressed_data_size;

    // if there is repetition or definition level data add RLE encoding
    auto const rle_bits =
      ck_g[warp_id].col_desc->num_def_level_bits() + ck_g[warp_id].col_desc->num_rep_level_bits();
    if (rle_bits > 0) { encodings |= encoding_to_mask(Encoding::RLE); }
    chunks[chunk_id].encodings = encodings;
  }
}

/**
 * Minimal thrift compact protocol support
 */
inline __device__ uint8_t* cpw_put_uint8(uint8_t* p, uint8_t v)
{
  *p++ = v;
  return p;
}

inline __device__ uint8_t* cpw_put_uint32(uint8_t* p, uint32_t v)
{
  while (v > 0x7f) {
    *p++ = v | 0x80;
    v >>= 7;
  }
  *p++ = v;
  return p;
}

inline __device__ uint8_t* cpw_put_uint64(uint8_t* p, uint64_t v)
{
  while (v > 0x7f) {
    *p++ = v | 0x80;
    v >>= 7;
  }
  *p++ = v;
  return p;
}

inline __device__ uint8_t* cpw_put_int32(uint8_t* p, int32_t v)
{
  int32_t s = (v < 0);
  return cpw_put_uint32(p, (v ^ -s) * 2 + s);
}

inline __device__ uint8_t* cpw_put_int64(uint8_t* p, int64_t v)
{
  int64_t s = (v < 0);
  return cpw_put_uint64(p, (v ^ -s) * 2 + s);
}

inline __device__ uint8_t* cpw_put_fldh(uint8_t* p, int f, int cur, FieldType t)
{
  auto const t_num = static_cast<uint8_t>(t);
  if (f > cur && f <= cur + 15) {
    *p++ = ((f - cur) << 4) | t_num;
    return p;
  } else {
    *p++ = t_num;
    return cpw_put_int32(p, f);
  }
}

class header_encoder {
  uint8_t* current_header_ptr;
  int current_field_index;

 public:
  inline __device__ header_encoder(uint8_t* header_start)
    : current_header_ptr(header_start), current_field_index(0)
  {
  }

  inline __device__ void field_struct_begin(int field)
  {
    current_header_ptr =
      cpw_put_fldh(current_header_ptr, field, current_field_index, FieldType::STRUCT);
    current_field_index = 0;
  }

  inline __device__ void field_struct_end(int field)
  {
    *current_header_ptr++ = 0;
    current_field_index   = field;
  }

  inline __device__ void field_list_begin(int field, size_t len, FieldType type)
  {
    current_header_ptr =
      cpw_put_fldh(current_header_ptr, field, current_field_index, FieldType::LIST);
    auto const t_num   = static_cast<uint8_t>(type);
    current_header_ptr = cpw_put_uint8(
      current_header_ptr, static_cast<uint8_t>((std::min(len, size_t{0xfu}) << 4) | t_num));
    if (len >= 0xf) { current_header_ptr = cpw_put_uint32(current_header_ptr, len); }
    current_field_index = 0;
  }

  inline __device__ void field_list_end(int field) { current_field_index = field; }

  inline __device__ void put_bool(bool value)
  {
    auto const type_byte =
      static_cast<uint8_t>(value ? FieldType::BOOLEAN_TRUE : FieldType::BOOLEAN_FALSE);
    current_header_ptr = cpw_put_uint8(current_header_ptr, type_byte);
  }

  inline __device__ void put_binary(void const* value, uint32_t length)
  {
    current_header_ptr = cpw_put_uint32(current_header_ptr, length);
    memcpy(current_header_ptr, value, length);
    current_header_ptr += length;
  }

  template <typename T>
  inline __device__ void put_int64(T value)
  {
    current_header_ptr = cpw_put_int64(current_header_ptr, static_cast<int64_t>(value));
  }

  inline __device__ void field_bool(int field, bool value)
  {
    current_header_ptr  = cpw_put_fldh(current_header_ptr,
                                      field,
                                      current_field_index,
                                      value ? FieldType::BOOLEAN_TRUE : FieldType::BOOLEAN_FALSE);
    current_field_index = field;
  }

  template <typename T>
  inline __device__ void field_int32(int field, T value)
  {
    current_header_ptr =
      cpw_put_fldh(current_header_ptr, field, current_field_index, FieldType::I32);
    current_header_ptr  = cpw_put_int32(current_header_ptr, static_cast<int32_t>(value));
    current_field_index = field;
  }

  template <typename T>
  inline __device__ void field_int64(int field, T value)
  {
    current_header_ptr =
      cpw_put_fldh(current_header_ptr, field, current_field_index, FieldType::I64);
    current_header_ptr  = cpw_put_int64(current_header_ptr, static_cast<int64_t>(value));
    current_field_index = field;
  }

  inline __device__ void field_binary(int field, void const* value, uint32_t length)
  {
    current_header_ptr =
      cpw_put_fldh(current_header_ptr, field, current_field_index, FieldType::BINARY);
    current_header_ptr = cpw_put_uint32(current_header_ptr, length);
    memcpy(current_header_ptr, value, length);
    current_header_ptr += length;
    current_field_index = field;
  }

  inline __device__ void end(uint8_t** header_end, bool termination_flag = true)
  {
    if (not termination_flag) { *current_header_ptr++ = 0; }
    *header_end = current_header_ptr;
  }

  inline __device__ uint8_t* get_ptr() { return current_header_ptr; }

  inline __device__ void set_ptr(uint8_t* ptr) { current_header_ptr = ptr; }
};

namespace {

// byteswap 128 bit integer, placing result in dst in network byte order.
// dst must point to at least 16 bytes of memory.
__device__ void byte_reverse128(__int128_t v, void* dst)
{
  auto const v_char_ptr = reinterpret_cast<unsigned char const*>(&v);
  auto const d_char_ptr = static_cast<unsigned char*>(dst);
  thrust::copy(thrust::seq,
               thrust::make_reverse_iterator(v_char_ptr + sizeof(v)),
               thrust::make_reverse_iterator(v_char_ptr),
               d_char_ptr);
}

/**
 * @brief Test to see if a span contains all valid UTF-8 characters.
 *
 * @param span device_span to test.
 * @return true if the span contains all valid UTF-8 characters.
 */
__device__ bool is_valid_utf8(device_span<unsigned char const> span)
{
  auto idx = 0;
  while (idx < span.size_bytes()) {
    // UTF-8 character should start with valid beginning bit pattern
    if (not strings::detail::is_valid_begin_utf8_char(span[idx])) { return false; }
    // subsequent elements of the character should be continuation chars
    auto const width = strings::detail::bytes_in_utf8_byte(span[idx++]);
    for (size_type i = 1; i < width && idx < span.size_bytes(); i++, idx++) {
      if (not strings::detail::is_utf8_continuation_char(span[idx])) { return false; }
    }
  }

  return true;
}

/**
 * @brief Increment part of a UTF-8 character.
 *
 * Attempt to increment the char pointed to by ptr, which is assumed to be part of a valid UTF-8
 * character. Returns true if successful, false if the increment caused an overflow, in which case
 * the data at ptr will be set to the lowest valid UTF-8 bit pattern (start or continuation).
 * Will halt execution if passed invalid UTF-8.
 */
__device__ bool increment_utf8_at(unsigned char* ptr)
{
  unsigned char elem = *ptr;
  // elem is one of (no 5 or 6 byte chars allowed):
  //  0b0vvvvvvv a 1 byte character
  //  0b10vvvvvv a continuation byte
  //  0b110vvvvv start of a 2 byte character
  //  0b1110vvvv start of a 3 byte character
  //  0b11110vvv start of a 4 byte character

  // TODO(ets): starting at 4 byte and working down.  Should probably start low and work higher.
  uint8_t mask  = 0xF8;
  uint8_t valid = 0xF0;

  while (mask != 0) {
    if ((elem & mask) == valid) {
      elem++;
      if ((elem & mask) != mask) {  // no overflow
        *ptr = elem;
        return true;
      }
      *ptr = valid;
      return false;
    }
    mask <<= 1;
    valid <<= 1;
  }

  // should not reach here since we test for valid UTF-8 higher up the call chain
  CUDF_UNREACHABLE("Trying to increment non-utf8");
}

/**
 * @brief Attempt to truncate a span of UTF-8 characters to at most truncate_length_bytes.
 *
 * If is_min is false, then the final character (or characters if there is overflow) will be
 * incremented so that the resultant UTF-8 will still be a valid maximum. scratch is only used when
 * is_min is false, and must be at least truncate_length bytes in size. If the span cannot be
 * truncated, leave it untouched and return the original length.
 *
 * @return Pair object containing a pointer to the truncated data and its length.
 */
__device__ std::pair<void const*, uint32_t> truncate_utf8(device_span<unsigned char const> span,
                                                          bool is_min,
                                                          void* scratch,
                                                          int32_t truncate_length)
{
  // we know at this point that truncate_length < size_bytes, so
  // there is data at [len]. work backwards until we find
  // the start of a UTF-8 encoded character, since UTF-8 characters may be multi-byte.
  auto len = truncate_length;
  while (not strings::detail::is_begin_utf8_char(span[len]) && len > 0) {
    len--;
  }

  if (len != 0) {
    if (is_min) { return {span.data(), len}; }
    memcpy(scratch, span.data(), len);
    // increment last byte, working backwards if the byte overflows
    auto const ptr = static_cast<unsigned char*>(scratch);
    for (int32_t i = len - 1; i >= 0; i--) {
      if (increment_utf8_at(&ptr[i])) {  // true if no overflow
        return {scratch, len};
      }
    }
    // cannot increment, so fall through
  }

  // couldn't truncate, return original value
  return {span.data(), span.size_bytes()};
}

/**
 * @brief Attempt to truncate a span of binary data to at most truncate_length bytes.
 *
 * If is_min is false, then the final byte (or bytes if there is overflow) will be
 * incremented so that the resultant binary will still be a valid maximum. scratch is only used when
 * is_min is false, and must be at least truncate_length bytes in size. If the span cannot be
 * truncated, leave it untouched and return the original length.
 *
 * @return Pair object containing a pointer to the truncated data and its length.
 */
__device__ std::pair<void const*, uint32_t> truncate_binary(device_span<uint8_t const> arr,
                                                            bool is_min,
                                                            void* scratch,
                                                            int32_t truncate_length)
{
  if (is_min) { return {arr.data(), truncate_length}; }
  memcpy(scratch, arr.data(), truncate_length);
  // increment last byte, working backwards if the byte overflows
  auto const ptr = static_cast<uint8_t*>(scratch);
  for (int32_t i = truncate_length - 1; i >= 0; i--) {
    ptr[i]++;
    if (ptr[i] != 0) {  // no overflow
      return {scratch, i + 1};
    }
  }

  // couldn't truncate, return original value
  return {arr.data(), arr.size_bytes()};
}

// TODO (ets): the assumption here is that string columns might have UTF-8 or plain binary,
// while binary columns are assumed to be binary and will be treated as such.  If this assumption
// is incorrect, then truncate_byte_array() and truncate_string() should just be combined into
// a single function.
/**
 * @brief Attempt to truncate a UTF-8 string to at most truncate_length bytes.
 */
__device__ std::pair<void const*, uint32_t> truncate_string(string_view const& str,
                                                            bool is_min,
                                                            void* scratch,
                                                            int32_t truncate_length)
{
  if (truncate_length == NO_TRUNC_STATS or str.size_bytes() <= truncate_length) {
    return {str.data(), str.size_bytes()};
  }

  // convert char to unsigned since UTF-8 is just bytes, not chars.  can't use std::byte because
  // that can't be incremented.
  auto const span = device_span<unsigned char const>(
    reinterpret_cast<unsigned char const*>(str.data()), str.size_bytes());

  // if str is all 8-bit chars, or is actually not UTF-8, then we can just use truncate_binary()
  if (str.size_bytes() != str.length() and is_valid_utf8(span.first(truncate_length))) {
    return truncate_utf8(span, is_min, scratch, truncate_length);
  }
  return truncate_binary(span, is_min, scratch, truncate_length);
}

/**
 * @brief Attempt to truncate a binary array to at most truncate_length bytes.
 */
__device__ std::pair<void const*, uint32_t> truncate_byte_array(
  statistics::byte_array_view const& arr, bool is_min, void* scratch, int32_t truncate_length)
{
  if (truncate_length == NO_TRUNC_STATS or arr.size_bytes() <= truncate_length) {
    return {arr.data(), arr.size_bytes()};
  }

  // convert std::byte to uint8_t since bytes can't be incremented
  device_span<uint8_t const> const span{reinterpret_cast<uint8_t const*>(arr.data()),
                                        arr.size_bytes()};
  return truncate_binary(span, is_min, scratch, truncate_length);
}

/**
 * @brief Find a min or max value of the proper form to be included in Parquet statistics
 * structures.
 *
 * Given a statistics_val union and a data type, perform any transformations needed to produce a
 * valid min or max binary value.  String and byte array types will be truncated if they exceed
 * truncate_length.
 */
__device__ std::pair<void const*, uint32_t> get_extremum(statistics_val const* stats_val,
                                                         statistics_dtype dtype,
                                                         void* scratch,
                                                         bool is_min,
                                                         int32_t truncate_length)
{
  switch (dtype) {
    case dtype_bool: return {stats_val, sizeof(bool)};
    case dtype_int8:
    case dtype_int16:
    case dtype_int32:
    case dtype_date32: return {stats_val, sizeof(int32_t)};
    case dtype_float32: {
      auto const fp_scratch = static_cast<float*>(scratch);
      fp_scratch[0]         = stats_val->fp_val;
      return {scratch, sizeof(float)};
    }
    case dtype_int64:
    case dtype_decimal64:
    case dtype_timestamp64:
    case dtype_float64: return {stats_val, sizeof(int64_t)};
    case dtype_decimal128:
      byte_reverse128(stats_val->d128_val, scratch);
      return {scratch, sizeof(__int128_t)};
    case dtype_string: return truncate_string(stats_val->str_val, is_min, scratch, truncate_length);
    case dtype_byte_array:
      return truncate_byte_array(stats_val->byte_val, is_min, scratch, truncate_length);
    default: CUDF_UNREACHABLE("Invalid statistics data type");
  }
}

}  // namespace

__device__ uint8_t* EncodeStatistics(uint8_t* start,
                                     statistics_chunk const* s,
                                     statistics_dtype dtype,
                                     void* scratch)
{
  uint8_t* end;
  header_encoder encoder(start);
  encoder.field_int64(3, s->null_count);
  if (s->has_minmax) {
    auto const [max_ptr, max_size] =
      get_extremum(&s->max_value, dtype, scratch, false, NO_TRUNC_STATS);
    encoder.field_binary(5, max_ptr, max_size);
    auto const [min_ptr, min_size] =
      get_extremum(&s->min_value, dtype, scratch, true, NO_TRUNC_STATS);
    encoder.field_binary(6, min_ptr, min_size);
  }
  encoder.end(&end);
  return end;
}

// blockDim(128, 1, 1)
CUDF_KERNEL void __launch_bounds__(128)
  gpuEncodePageHeaders(device_span<EncPage> pages,
                       device_span<compression_result const> comp_results,
                       device_span<statistics_chunk const> page_stats,
                       statistics_chunk const* chunk_stats)
{
  // When this whole kernel becomes single thread, the following variables need not be __shared__
  __shared__ __align__(8) parquet_column_device_view col_g;
  __shared__ __align__(8) EncColumnChunk ck_g;
  __shared__ __align__(8) EncPage page_g;
  __shared__ __align__(8) unsigned char scratch[MIN_STATS_SCRATCH_SIZE];

  auto const t = threadIdx.x;

  if (t == 0) {
    uint8_t *hdr_start, *hdr_end;
    uint32_t compressed_page_size, uncompressed_page_size;

    page_g = pages[blockIdx.x];
    ck_g   = *page_g.chunk;
    col_g  = *ck_g.col_desc;

    if (chunk_stats && &pages[blockIdx.x] == ck_g.pages) {  // Is this the first page in a chunk?
      hdr_start = (ck_g.is_compressed) ? ck_g.compressed_bfr : ck_g.uncompressed_bfr;
      hdr_end =
        EncodeStatistics(hdr_start, &chunk_stats[page_g.chunk_id], col_g.stats_dtype, scratch);
      page_g.chunk->ck_stat_size = static_cast<uint32_t>(hdr_end - hdr_start);
    }
    uncompressed_page_size = page_g.data_size;
    if (ck_g.is_compressed) {
      auto const lvl_bytes = page_g.is_v2() ? page_g.level_bytes() : 0;
      hdr_start            = page_g.compressed_data;
      compressed_page_size =
        static_cast<uint32_t>(comp_results[blockIdx.x].bytes_written) + lvl_bytes;
      page_g.comp_data_size = compressed_page_size;
    } else {
      hdr_start            = page_g.page_data;
      compressed_page_size = uncompressed_page_size;
    }
    header_encoder encoder(hdr_start);
    PageType page_type = page_g.page_type;

    encoder.field_int32(1, page_type);
    encoder.field_int32(2, uncompressed_page_size);
    encoder.field_int32(3, compressed_page_size);

    if (page_type == PageType::DATA_PAGE) {
      // DataPageHeader
      encoder.field_struct_begin(5);
      encoder.field_int32(1, page_g.num_values);  // NOTE: num_values != num_rows for list types
      encoder.field_int32(2, page_g.encoding);    // encoding
      encoder.field_int32(3, Encoding::RLE);      // definition_level_encoding
      encoder.field_int32(4, Encoding::RLE);      // repetition_level_encoding
      // Optionally encode page-level statistics
      if (not page_stats.empty()) {
        encoder.field_struct_begin(5);
        encoder.set_ptr(
          EncodeStatistics(encoder.get_ptr(), &page_stats[blockIdx.x], col_g.stats_dtype, scratch));
        encoder.field_struct_end(5);
      }
      encoder.field_struct_end(5);
    } else if (page_type == PageType::DATA_PAGE_V2) {
      // DataPageHeaderV2
      encoder.field_struct_begin(8);
      encoder.field_int32(1, page_g.num_values);
      encoder.field_int32(2, page_g.num_nulls);
      encoder.field_int32(3, page_g.num_rows);
      encoder.field_int32(4, page_g.encoding);
      encoder.field_int32(5, page_g.def_lvl_bytes);
      encoder.field_int32(6, page_g.rep_lvl_bytes);
      encoder.field_bool(7, ck_g.is_compressed);  // TODO can compress at page level now
      // Optionally encode page-level statistics
      if (not page_stats.empty()) {
        encoder.field_struct_begin(8);
        encoder.set_ptr(
          EncodeStatistics(encoder.get_ptr(), &page_stats[blockIdx.x], col_g.stats_dtype, scratch));
        encoder.field_struct_end(8);
      }
      encoder.field_struct_end(8);
    } else {
      // DictionaryPageHeader
      encoder.field_struct_begin(7);
      encoder.field_int32(1, ck_g.num_dict_entries);  // number of values in dictionary
      encoder.field_int32(2, page_g.encoding);
      encoder.field_struct_end(7);
    }
    encoder.end(&hdr_end, false);
    page_g.hdr_size = (uint32_t)(hdr_end - hdr_start);
  }
  __syncthreads();
  if (t == 0) pages[blockIdx.x] = page_g;
}

// blockDim(1024, 1, 1)
CUDF_KERNEL void __launch_bounds__(1024)
  gpuGatherPages(device_span<EncColumnChunk> chunks, device_span<EncPage const> pages)
{
  __shared__ __align__(8) EncColumnChunk ck_g;
  __shared__ __align__(8) EncPage page_g;

  auto const t = threadIdx.x;
  uint8_t *dst, *dst_base;
  EncPage const* first_page;
  uint32_t num_pages, uncompressed_size;

  if (t == 0) ck_g = chunks[blockIdx.x];
  __syncthreads();

  first_page = ck_g.pages;
  num_pages  = ck_g.num_pages;
  dst        = (ck_g.is_compressed) ? ck_g.compressed_bfr : ck_g.uncompressed_bfr;
  dst += ck_g.ck_stat_size;  // Skip over chunk statistics
  dst_base          = dst;
  uncompressed_size = ck_g.bfr_size;
  for (uint32_t page = 0; page < num_pages; page++) {
    uint8_t const* src;
    uint32_t hdr_len, data_len;

    if (t == 0) { page_g = first_page[page]; }
    __syncthreads();

    src = ck_g.is_compressed ? page_g.compressed_data : page_g.page_data;
    // Copy page header
    hdr_len = page_g.hdr_size;
    memcpy_block<1024, true>(dst, src, hdr_len, t);
    src += page_g.max_hdr_size;
    dst += hdr_len;
    uncompressed_size += hdr_len;
    data_len = ck_g.is_compressed ? page_g.comp_data_size : page_g.data_size;
    // Copy page data. For V2, the level data and page data are disjoint.
    if (page_g.is_v2()) {
      auto const lvl_len = page_g.level_bytes();
      memcpy_block<1024, true>(dst, src, lvl_len, t);
      src += page_g.max_lvl_size;
      dst += lvl_len;
      data_len -= lvl_len;
    }
    memcpy_block<1024, true>(dst, src, data_len, t);
    dst += data_len;
    __syncthreads();
    if (t == 0 && page == 0 && ck_g.use_dictionary) { ck_g.dictionary_size = hdr_len + data_len; }
  }
  if (t == 0) {
    chunks[blockIdx.x].bfr_size        = uncompressed_size;
    chunks[blockIdx.x].compressed_size = (dst - dst_base);
    if (ck_g.use_dictionary) { chunks[blockIdx.x].dictionary_size = ck_g.dictionary_size; }
  }
}

namespace {

/**
 * @brief Tests if statistics are comparable given the column's
 * physical and converted types
 */
__device__ bool is_comparable(Type ptype, ConvertedType ctype)
{
  switch (ptype) {
    case Type::BOOLEAN:
    case Type::INT32:
    case Type::INT64:
    case Type::FLOAT:
    case Type::DOUBLE:
    case Type::BYTE_ARRAY: return true;
    case Type::FIXED_LEN_BYTE_ARRAY:
      if (ctype == ConvertedType::DECIMAL) { return true; }
      [[fallthrough]];
    default: return false;
  }
}

/**
 * @brief Compares two values.
 * @return -1 if v1 < v2, 0 if v1 == v2, 1 if v1 > v2
 */
template <typename T>
constexpr __device__ int32_t compare(T& v1, T& v2)
{
  return (v1 > v2) - (v1 < v2);
}

/**
 * @brief Compares two statistics_val structs.
 * @return < 0 if v1 < v2, 0 if v1 == v2, > 0 if v1 > v2
 */
__device__ int32_t compare_values(Type ptype,
                                  ConvertedType ctype,
                                  statistics_val const& v1,
                                  statistics_val const& v2)
{
  switch (ptype) {
    case Type::BOOLEAN: return compare(v1.u_val, v2.u_val);
    case Type::INT32:
    case Type::INT64:
      switch (ctype) {
        case ConvertedType::UINT_8:
        case ConvertedType::UINT_16:
        case ConvertedType::UINT_32:
        case ConvertedType::UINT_64: return compare(v1.u_val, v2.u_val);
        default:  // assume everything else is signed
          return compare(v1.i_val, v2.i_val);
      }
    case Type::FLOAT:
    case Type::DOUBLE: return compare(v1.fp_val, v2.fp_val);
    case Type::BYTE_ARRAY: return static_cast<string_view>(v1.str_val).compare(v2.str_val);
    case Type::FIXED_LEN_BYTE_ARRAY:
      if (ctype == ConvertedType::DECIMAL) { return compare(v1.d128_val, v2.d128_val); }
  }
  // calling is_comparable() should prevent reaching here
  CUDF_UNREACHABLE("Trying to compare non-comparable type");
  return 0;
}

/**
 * @brief Determine if a set of statstistics are in ascending order.
 */
__device__ bool is_ascending(statistics_chunk const* s,
                             Type ptype,
                             ConvertedType ctype,
                             uint32_t num_pages)
{
  for (uint32_t i = 1; i < num_pages; i++) {
    if (compare_values(ptype, ctype, s[i - 1].min_value, s[i].min_value) > 0 ||
        compare_values(ptype, ctype, s[i - 1].max_value, s[i].max_value) > 0) {
      return false;
    }
  }
  return true;
}

/**
 * @brief Determine if a set of statstistics are in descending order.
 */
__device__ bool is_descending(statistics_chunk const* s,
                              Type ptype,
                              ConvertedType ctype,
                              uint32_t num_pages)
{
  for (uint32_t i = 1; i < num_pages; i++) {
    if (compare_values(ptype, ctype, s[i - 1].min_value, s[i].min_value) < 0 ||
        compare_values(ptype, ctype, s[i - 1].max_value, s[i].max_value) < 0) {
      return false;
    }
  }
  return true;
}

/**
 * @brief Determine the ordering of a set of statistics.
 */
__device__ int32_t calculate_boundary_order(statistics_chunk const* s,
                                            Type ptype,
                                            ConvertedType ctype,
                                            uint32_t num_pages)
{
  if (not is_comparable(ptype, ctype)) { return BoundaryOrder::UNORDERED; }
  if (is_ascending(s, ptype, ctype, num_pages)) {
    return BoundaryOrder::ASCENDING;
  } else if (is_descending(s, ptype, ctype, num_pages)) {
    return BoundaryOrder::DESCENDING;
  }
  return BoundaryOrder::UNORDERED;
}

// align ptr to an 8-byte boundary. address returned will be <= ptr.
constexpr __device__ void* align8(void* ptr)
{
  // it's ok to round down because we have an extra 7 bytes in the buffer
  auto algn = 3 & reinterpret_cast<std::uintptr_t>(ptr);
  return static_cast<char*>(ptr) - algn;
}

struct mask_tform {
  __device__ uint32_t operator()(EncPage const& p) { return static_cast<uint32_t>(p.kernel_mask); }
};

}  // namespace

// blockDim(1, 1, 1)
CUDF_KERNEL void __launch_bounds__(1)
  gpuEncodeColumnIndexes(device_span<EncColumnChunk> chunks,
                         device_span<statistics_chunk const> column_stats,
                         int32_t column_index_truncate_length)
{
  __align__(8) unsigned char s_scratch[MIN_STATS_SCRATCH_SIZE];
  uint8_t* col_idx_end;

  if (column_stats.empty()) { return; }

  auto const ck_g                = &chunks[blockIdx.x];
  uint32_t const num_pages       = ck_g->num_pages;
  auto const& col_g              = *ck_g->col_desc;
  uint32_t const first_data_page = ck_g->use_dictionary ? 1 : 0;
  uint32_t const num_data_pages  = num_pages - first_data_page;
  uint32_t const pageidx         = ck_g->first_page;
  size_t var_bytes               = 0;

  header_encoder encoder(ck_g->column_index_blob);

  // make sure scratch is aligned properly. here column_index_size indicates
  // how much scratch space is available for this chunk, including space for
  // truncation scratch + padding for alignment.
  void* scratch =
    column_index_truncate_length < MIN_STATS_SCRATCH_SIZE
      ? s_scratch
      : align8(ck_g->column_index_blob + ck_g->column_index_size - column_index_truncate_length);

  // null_pages
  encoder.field_list_begin(1, num_data_pages, FieldType::BOOLEAN_TRUE);
  for (uint32_t page = first_data_page; page < num_pages; page++) {
    encoder.put_bool(column_stats[pageidx + page].non_nulls == 0);
  }
  encoder.field_list_end(1);
  // min_values
  encoder.field_list_begin(2, num_data_pages, FieldType::BINARY);
  for (uint32_t page = first_data_page; page < num_pages; page++) {
    auto const [min_ptr, min_size] = get_extremum(&column_stats[pageidx + page].min_value,
                                                  col_g.stats_dtype,
                                                  scratch,
                                                  true,
                                                  column_index_truncate_length);
    encoder.put_binary(min_ptr, min_size);
  }
  encoder.field_list_end(2);
  // max_values
  encoder.field_list_begin(3, num_data_pages, FieldType::BINARY);
  for (uint32_t page = first_data_page; page < num_pages; page++) {
    auto const [max_ptr, max_size] = get_extremum(&column_stats[pageidx + page].max_value,
                                                  col_g.stats_dtype,
                                                  scratch,
                                                  false,
                                                  column_index_truncate_length);
    encoder.put_binary(max_ptr, max_size);
  }
  encoder.field_list_end(3);
  // boundary_order
  encoder.field_int32(4,
                      calculate_boundary_order(&column_stats[first_data_page + pageidx],
                                               col_g.physical_type,
                                               col_g.converted_type,
                                               num_pages - first_data_page));
  // null_counts
  encoder.field_list_begin(5, num_data_pages, FieldType::I64);
  for (uint32_t page = first_data_page; page < num_pages; page++) {
    encoder.put_int64(column_stats[pageidx + page].null_count);
  }
  encoder.field_list_end(5);

  // find pointers to chunk histograms
  auto const cd          = ck_g->col_desc;
  auto const ck_def_hist = ck_g->def_histogram_data + (num_data_pages) * (cd->max_def_level + 1);
  auto const ck_rep_hist = ck_g->rep_histogram_data + (num_data_pages) * (cd->max_rep_level + 1);

  auto const page_start = ck_g->pages + first_data_page;
  auto const page_end   = ck_g->pages + ck_g->num_pages;

  // optionally encode histograms and sum var_bytes.
  if (cd->max_rep_level > REP_LVL_HIST_CUTOFF) {
    encoder.field_list_begin(6, num_data_pages * (cd->max_rep_level + 1), FieldType::I64);
    thrust::for_each(thrust::seq, page_start, page_end, [&] __device__(auto const& page) {
      for (int i = 0; i < cd->max_rep_level + 1; i++) {
        encoder.put_int64(page.rep_histogram[i]);
        ck_rep_hist[i] += page.rep_histogram[i];
      }
    });
    encoder.field_list_end(6);
  }

  if (cd->max_def_level > DEF_LVL_HIST_CUTOFF) {
    encoder.field_list_begin(7, num_data_pages * (cd->max_def_level + 1), FieldType::I64);
    thrust::for_each(thrust::seq, page_start, page_end, [&] __device__(auto const& page) {
      for (int i = 0; i < cd->max_def_level + 1; i++) {
        encoder.put_int64(page.def_histogram[i]);
        ck_def_hist[i] += page.def_histogram[i];
      }
    });
    encoder.field_list_end(7);
  }

  if (col_g.physical_type == BYTE_ARRAY) {
    thrust::for_each(thrust::seq, page_start, page_end, [&] __device__(auto const& page) {
      var_bytes += page.var_bytes_size;
    });
  }

  encoder.end(&col_idx_end, false);

  // now reset column_index_size to the actual size of the encoded column index blob
  ck_g->column_index_size = static_cast<uint32_t>(col_idx_end - ck_g->column_index_blob);
  ck_g->var_bytes_size    = var_bytes;
}

void InitRowGroupFragments(device_2dspan<PageFragment> frag,
                           device_span<parquet_column_device_view const> col_desc,
                           device_span<partition_info const> partitions,
                           device_span<int const> part_frag_offset,
                           uint32_t fragment_size,
                           rmm::cuda_stream_view stream)
{
  auto const num_columns              = frag.size().first;
  auto const num_fragments_per_column = frag.size().second;
  auto const grid_y = std::min(static_cast<uint32_t>(num_fragments_per_column), MAX_GRID_Y_SIZE);
  dim3 const dim_grid(num_columns, grid_y);  // 1 threadblock per fragment
  gpuInitRowGroupFragments<512><<<dim_grid, 512, 0, stream.value()>>>(
    frag, col_desc, partitions, part_frag_offset, fragment_size);
}

void CalculatePageFragments(device_span<PageFragment> frag,
                            device_span<size_type const> column_frag_sizes,
                            rmm::cuda_stream_view stream)
{
  gpuCalculatePageFragments<512><<<frag.size(), 512, 0, stream.value()>>>(frag, column_frag_sizes);
}

void InitFragmentStatistics(device_span<statistics_group> groups,
                            device_span<PageFragment const> fragments,
                            rmm::cuda_stream_view stream)
{
  int const num_fragments = fragments.size();
  int const dim =
    util::div_rounding_up_safe(num_fragments, encode_block_size / cudf::detail::warp_size);
  gpuInitFragmentStats<<<dim, encode_block_size, 0, stream.value()>>>(groups, fragments);
}

void InitEncoderPages(device_2dspan<EncColumnChunk> chunks,
                      device_span<EncPage> pages,
                      device_span<size_type> page_sizes,
                      device_span<size_type> comp_page_sizes,
                      device_span<parquet_column_device_view const> col_desc,
                      int32_t num_columns,
                      size_t max_page_size_bytes,
                      size_type max_page_size_rows,
                      uint32_t page_align,
                      bool write_v2_headers,
                      statistics_merge_group* page_grstats,
                      statistics_merge_group* chunk_grstats,
                      rmm::cuda_stream_view stream)
{
  auto num_rowgroups = chunks.size().first;
  dim3 dim_grid(num_columns, num_rowgroups);  // 1 threadblock per rowgroup
  gpuInitPages<<<dim_grid, encode_block_size, 0, stream.value()>>>(chunks,
                                                                   pages,
                                                                   page_sizes,
                                                                   comp_page_sizes,
                                                                   col_desc,
                                                                   page_grstats,
                                                                   chunk_grstats,
                                                                   num_columns,
                                                                   max_page_size_bytes,
                                                                   max_page_size_rows,
                                                                   page_align,
                                                                   write_v2_headers);
}

void EncodePages(device_span<EncPage> pages,
                 bool write_v2_headers,
                 device_span<device_span<uint8_t const>> comp_in,
                 device_span<device_span<uint8_t>> comp_out,
                 device_span<compression_result> comp_results,
                 rmm::cuda_stream_view stream)
{
  auto num_pages = pages.size();

  // determine which kernels to invoke
  auto mask_iter       = thrust::make_transform_iterator(pages.begin(), mask_tform{});
  uint32_t kernel_mask = thrust::reduce(
    rmm::exec_policy(stream), mask_iter, mask_iter + pages.size(), 0U, thrust::bit_or<uint32_t>{});

  // get the number of streams we need from the pool
  int nkernels = std::bitset<32>(kernel_mask).count();
  auto streams = cudf::detail::fork_streams(stream, nkernels);

  // A page is part of one column. This is launching 1 block per page. 1 block will exclusively
  // deal with one datatype.

  int s_idx = 0;
  if (BitAnd(kernel_mask, encode_kernel_mask::PLAIN) != 0) {
    auto const strm = streams[s_idx++];
    gpuEncodePageLevels<encode_block_size><<<num_pages, encode_block_size, 0, strm.value()>>>(
      pages, write_v2_headers, encode_kernel_mask::PLAIN);
    gpuEncodePages<encode_block_size><<<num_pages, encode_block_size, 0, strm.value()>>>(
      pages, comp_in, comp_out, comp_results, write_v2_headers);
  }
  if (BitAnd(kernel_mask, encode_kernel_mask::DELTA_BINARY) != 0) {
    auto const strm = streams[s_idx++];
    gpuEncodePageLevels<encode_block_size><<<num_pages, encode_block_size, 0, strm.value()>>>(
      pages, write_v2_headers, encode_kernel_mask::DELTA_BINARY);
    gpuEncodeDeltaBinaryPages<encode_block_size>
      <<<num_pages, encode_block_size, 0, strm.value()>>>(pages, comp_in, comp_out, comp_results);
  }
  if (BitAnd(kernel_mask, encode_kernel_mask::DELTA_LENGTH_BA) != 0) {
    auto const strm = streams[s_idx++];
    gpuEncodePageLevels<encode_block_size><<<num_pages, encode_block_size, 0, strm.value()>>>(
      pages, write_v2_headers, encode_kernel_mask::DELTA_LENGTH_BA);
    gpuEncodeDeltaLengthByteArrayPages<encode_block_size>
      <<<num_pages, encode_block_size, 0, strm.value()>>>(pages, comp_in, comp_out, comp_results);
  }
  if (BitAnd(kernel_mask, encode_kernel_mask::DELTA_BYTE_ARRAY) != 0) {
    auto const strm = streams[s_idx++];
    gpuEncodePageLevels<encode_block_size><<<num_pages, encode_block_size, 0, strm.value()>>>(
      pages, write_v2_headers, encode_kernel_mask::DELTA_BYTE_ARRAY);
    gpuEncodeDeltaByteArrayPages<encode_block_size>
      <<<num_pages, encode_block_size, 0, strm.value()>>>(pages, comp_in, comp_out, comp_results);
  }
  if (BitAnd(kernel_mask, encode_kernel_mask::DICTIONARY) != 0) {
    auto const strm = streams[s_idx++];
    gpuEncodePageLevels<encode_block_size><<<num_pages, encode_block_size, 0, strm.value()>>>(
      pages, write_v2_headers, encode_kernel_mask::DICTIONARY);
    gpuEncodeDictPages<encode_block_size><<<num_pages, encode_block_size, 0, strm.value()>>>(
      pages, comp_in, comp_out, comp_results, write_v2_headers);
  }

  cudf::detail::join_streams(streams, stream);
}

void DecideCompression(device_span<EncColumnChunk> chunks, rmm::cuda_stream_view stream)
{
  auto const num_blocks =
    util::div_rounding_up_safe<int>(chunks.size(), decide_compression_warps_in_block);
  gpuDecideCompression<<<num_blocks, decide_compression_block_size, 0, stream.value()>>>(chunks);
}

void EncodePageHeaders(device_span<EncPage> pages,
                       device_span<compression_result const> comp_results,
                       device_span<statistics_chunk const> page_stats,
                       statistics_chunk const* chunk_stats,
                       rmm::cuda_stream_view stream)
{
  // TODO: single thread task. No need for 128 threads/block. Earlier it used to employ rest of the
  // threads to coop load structs
  gpuEncodePageHeaders<<<pages.size(), encode_block_size, 0, stream.value()>>>(
    pages, comp_results, page_stats, chunk_stats);
}

void GatherPages(device_span<EncColumnChunk> chunks,
                 device_span<EncPage const> pages,
                 rmm::cuda_stream_view stream)
{
  gpuGatherPages<<<chunks.size(), 1024, 0, stream.value()>>>(chunks, pages);
}

void EncodeColumnIndexes(device_span<EncColumnChunk> chunks,
                         device_span<statistics_chunk const> column_stats,
                         int32_t column_index_truncate_length,
                         rmm::cuda_stream_view stream)
{
  gpuEncodeColumnIndexes<<<chunks.size(), 1, 0, stream.value()>>>(
    chunks, column_stats, column_index_truncate_length);
}

}  // namespace cudf::io::parquet::detail
