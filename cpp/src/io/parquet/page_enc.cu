#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019-2022, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "parquet_gpu.cuh"

#include <io/utilities/block_utils.cuh>

#include <cudf/detail/iterator.cuh>
#include <cudf/detail/utilities/assert.cuh>
#include <cudf/detail/utilities/cuda.cuh>
#include <cudf/detail/utilities/vector_factories.hpp>

#include <rmm/cuda_stream_view.hpp>
#include <rmm/exec_policy.hpp>

#include <hipcub/hipcub.hpp>

#include <cuda/std/chrono>

#include <thrust/binary_search.h>
#include <thrust/gather.h>
#include <thrust/iterator/discard_iterator.h>
#include <thrust/iterator/reverse_iterator.h>
#include <thrust/iterator/transform_iterator.h>
#include <thrust/iterator/zip_iterator.h>
#include <thrust/merge.h>
#include <thrust/scan.h>
#include <thrust/scatter.h>
#include <thrust/tuple.h>

namespace cudf {
namespace io {
namespace parquet {
namespace gpu {

namespace {
// Spark doesn't support RLE encoding for BOOLEANs
#ifdef ENABLE_BOOL_RLE
constexpr bool enable_bool_rle = true;
#else
constexpr bool enable_bool_rle = false;
#endif

using ::cudf::detail::device_2dspan;

constexpr uint32_t rle_buffer_size = (1 << 9);

// do not truncate statistics
constexpr int32_t NO_TRUNC_STATS = 0;

// minimum scratch space required for encoding statistics
constexpr size_t MIN_STATS_SCRATCH_SIZE = sizeof(__int128_t);

// mask to determine lane id
constexpr uint32_t WARP_MASK = cudf::detail::warp_size - 1;

// currently 64k - 1
constexpr uint32_t MAX_GRID_Y_SIZE = (1 << 16) - 1;

struct frag_init_state_s {
  parquet_column_device_view col;
  PageFragment frag;
};

struct page_enc_state_s {
  uint8_t* cur;          //!< current output ptr
  uint8_t* rle_out;      //!< current RLE write ptr
  uint32_t rle_run;      //!< current RLE run
  uint32_t run_val;      //!< current RLE run value
  uint32_t rle_pos;      //!< RLE encoder positions
  uint32_t rle_numvals;  //!< RLE input value count
  uint32_t rle_lit_count;
  uint32_t rle_rpt_count;
  uint32_t page_start_val;
  uint32_t chunk_start_val;
  volatile uint32_t rpt_map[4];
  volatile uint32_t scratch_red[32];
  EncPage page;
  EncColumnChunk ck;
  parquet_column_device_view col;
  uint32_t vals[rle_buffer_size];
};

/**
 * @brief Returns the size of the type in the Parquet file.
 */
constexpr uint32_t physical_type_len(Type physical_type, type_id id)
{
  if (physical_type == FIXED_LEN_BYTE_ARRAY and id == type_id::DECIMAL128) {
    return sizeof(__int128_t);
  }
  switch (physical_type) {
    case INT96: return 12u;
    case INT64:
    case DOUBLE: return sizeof(int64_t);
    case BOOLEAN: return 1u;
    default: return sizeof(int32_t);
  }
}

constexpr uint32_t max_RLE_page_size(uint8_t value_bit_width, uint32_t num_values)
{
  if (value_bit_width == 0) return 0;

  // Run length = 4, max(rle/bitpack header) = 5, add one byte per 256 values for overhead
  return 4 + 5 + util::div_rounding_up_unsafe(num_values * value_bit_width, 8) + (num_values / 256);
}

// subtract b from a, but return 0 if this would underflow
constexpr size_t underflow_safe_subtract(size_t a, size_t b)
{
  if (b > a) { return 0; }
  return a - b;
}

}  // anonymous namespace

// blockDim {512,1,1}
template <int block_size>
__global__ void __launch_bounds__(block_size)
  gpuInitPageFragments(device_2dspan<PageFragment> frag,
                       device_span<parquet_column_device_view const> col_desc,
                       device_span<partition_info const> partitions,
                       device_span<int const> part_frag_offset,
                       uint32_t fragment_size)
{
  __shared__ __align__(16) frag_init_state_s state_g;

  using block_reduce = hipcub::BlockReduce<uint32_t, block_size>;
  __shared__ typename block_reduce::TempStorage reduce_storage;

  frag_init_state_s* const s              = &state_g;
  uint32_t const t                        = threadIdx.x;
  auto const physical_type                = col_desc[blockIdx.x].physical_type;
  uint32_t const num_fragments_per_column = frag.size().second;

  if (t == 0) { s->col = col_desc[blockIdx.x]; }
  __syncthreads();

  auto const leaf_type = s->col.leaf_column->type().id();
  auto const dtype_len = physical_type_len(physical_type, leaf_type);

  for (uint32_t frag_y = blockIdx.y; frag_y < num_fragments_per_column; frag_y += gridDim.y) {
    if (t == 0) {
      // Find which partition this fragment came from
      auto it =
        thrust::upper_bound(thrust::seq, part_frag_offset.begin(), part_frag_offset.end(), frag_y);
      int p             = it - part_frag_offset.begin() - 1;
      int part_end_row  = partitions[p].start_row + partitions[p].num_rows;
      s->frag.start_row = (frag_y - part_frag_offset[p]) * fragment_size + partitions[p].start_row;

      // frag.num_rows = fragment_size except for the last fragment in partition which can be
      // smaller. num_rows is fixed but fragment size could be larger if the data is strings or
      // nested.
      s->frag.num_rows           = min(fragment_size, part_end_row - s->frag.start_row);
      s->frag.num_dict_vals      = 0;
      s->frag.fragment_data_size = 0;
      s->frag.dict_data_size     = 0;

      s->frag.start_value_idx = row_to_value_idx(s->frag.start_row, s->col);
      size_type end_value_idx = row_to_value_idx(s->frag.start_row + s->frag.num_rows, s->col);
      s->frag.num_leaf_values = end_value_idx - s->frag.start_value_idx;

      if (s->col.level_offsets != nullptr) {
        // For nested schemas, the number of values in a fragment is not directly related to the
        // number of encoded data elements or the number of rows.  It is simply the number of
        // repetition/definition values which together encode validity and nesting information.
        size_type first_level_val_idx = s->col.level_offsets[s->frag.start_row];
        size_type last_level_val_idx  = s->col.level_offsets[s->frag.start_row + s->frag.num_rows];
        s->frag.num_values            = last_level_val_idx - first_level_val_idx;
      } else {
        s->frag.num_values = s->frag.num_rows;
      }
    }
    __syncthreads();

    size_type nvals           = s->frag.num_leaf_values;
    size_type start_value_idx = s->frag.start_value_idx;

    for (uint32_t i = 0; i < nvals; i += block_size) {
      uint32_t val_idx  = start_value_idx + i + t;
      uint32_t is_valid = (i + t < nvals && val_idx < s->col.leaf_column->size())
                            ? s->col.leaf_column->is_valid(val_idx)
                            : 0;
      uint32_t len;
      if (is_valid) {
        len = dtype_len;
        if (physical_type == BYTE_ARRAY) {
          switch (leaf_type) {
            case type_id::STRING: {
              auto str = s->col.leaf_column->element<string_view>(val_idx);
              len += str.size_bytes();
            } break;
            case type_id::LIST: {
              auto list_element =
                get_element<statistics::byte_array_view>(*s->col.leaf_column, val_idx);
              len += list_element.size_bytes();
            } break;
            default: CUDF_UNREACHABLE("Unsupported data type for leaf column");
          }
        }
      } else {
        len = 0;
      }

      len = block_reduce(reduce_storage).Sum(len);
      if (t == 0) { s->frag.fragment_data_size += len; }
      __syncthreads();
      // page fragment size must fit in a 32-bit signed integer
      if (s->frag.fragment_data_size > std::numeric_limits<int32_t>::max()) {
        CUDF_UNREACHABLE("page fragment size exceeds maximum for i32");
      }
    }
    __syncthreads();
    if (t == 0) { frag[blockIdx.x][frag_y] = s->frag; }
  }
}

// blockDim {128,1,1}
__global__ void __launch_bounds__(128)
  gpuInitFragmentStats(device_2dspan<statistics_group> groups,
                       device_2dspan<PageFragment const> fragments,
                       device_span<parquet_column_device_view const> col_desc)
{
  uint32_t const lane_id                  = threadIdx.x & WARP_MASK;
  uint32_t const column_id                = blockIdx.x;
  uint32_t const num_fragments_per_column = fragments.size().second;

  uint32_t frag_id = blockIdx.y * 4 + (threadIdx.x / cudf::detail::warp_size);
  while (frag_id < num_fragments_per_column) {
    if (lane_id == 0) {
      statistics_group g;
      g.col                      = &col_desc[column_id];
      g.start_row                = fragments[column_id][frag_id].start_value_idx;
      g.num_rows                 = fragments[column_id][frag_id].num_leaf_values;
      groups[column_id][frag_id] = g;
    }
    frag_id += gridDim.y * 4;
  }
}

// blockDim {128,1,1}
__global__ void __launch_bounds__(128)
  gpuInitPages(device_2dspan<EncColumnChunk> chunks,
               device_span<gpu::EncPage> pages,
               device_span<size_type> page_sizes,
               device_span<size_type> comp_page_sizes,
               device_span<parquet_column_device_view const> col_desc,
               statistics_merge_group* page_grstats,
               statistics_merge_group* chunk_grstats,
               int32_t num_columns,
               size_t max_page_size_bytes,
               size_type max_page_size_rows,
               uint32_t page_align)
{
  // TODO: All writing seems to be done by thread 0. Could be replaced by thrust foreach
  __shared__ __align__(8) parquet_column_device_view col_g;
  __shared__ __align__(8) EncColumnChunk ck_g;
  __shared__ __align__(8) PageFragment frag_g;
  __shared__ __align__(8) EncPage page_g;
  __shared__ __align__(8) statistics_merge_group pagestats_g;

  uint32_t t = threadIdx.x;

  if (t == 0) {
    col_g  = col_desc[blockIdx.x];
    ck_g   = chunks[blockIdx.y][blockIdx.x];
    page_g = {};
  }
  __syncthreads();
  if (t < 32) {
    uint32_t fragments_in_chunk  = 0;
    uint32_t rows_in_page        = 0;
    uint32_t values_in_page      = 0;
    uint32_t leaf_values_in_page = 0;
    size_t page_size             = 0;
    uint32_t num_pages           = 0;
    uint32_t num_rows            = 0;
    uint32_t page_start          = 0;
    uint32_t page_offset         = ck_g.ck_stat_size;
    uint32_t num_dict_entries    = 0;
    uint32_t comp_page_offset    = ck_g.ck_stat_size;
    uint32_t page_headers_size   = 0;
    uint32_t max_page_data_size  = 0;
    uint32_t cur_row             = ck_g.start_row;
    uint32_t ck_max_stats_len    = 0;
    uint32_t max_stats_len       = 0;

    if (!t) {
      pagestats_g.col_dtype   = col_g.leaf_column->type();
      pagestats_g.stats_dtype = col_g.stats_dtype;
      pagestats_g.start_chunk = ck_g.first_fragment;
      pagestats_g.num_chunks  = 0;
    }
    if (ck_g.use_dictionary) {
      if (!t) {
        page_g.page_data       = ck_g.uncompressed_bfr + page_offset;
        page_g.compressed_data = ck_g.compressed_bfr + comp_page_offset;
        page_g.num_fragments   = 0;
        page_g.page_type       = PageType::DICTIONARY_PAGE;
        page_g.chunk           = &chunks[blockIdx.y][blockIdx.x];
        page_g.chunk_id        = blockIdx.y * num_columns + blockIdx.x;
        page_g.hdr_size        = 0;
        page_g.max_hdr_size    = 32;
        page_g.max_data_size   = ck_g.uniq_data_size;
        page_g.start_row       = cur_row;
        page_g.num_rows        = ck_g.num_dict_entries;
        page_g.num_leaf_values = ck_g.num_dict_entries;
        page_g.num_values      = ck_g.num_dict_entries;  // TODO: shouldn't matter for dict page
        page_offset +=
          util::round_up_unsafe(page_g.max_hdr_size + page_g.max_data_size, page_align);
        if (not comp_page_sizes.empty()) {
          comp_page_offset += page_g.max_hdr_size + comp_page_sizes[ck_g.first_page];
        }
        page_headers_size += page_g.max_hdr_size;
        max_page_data_size = max(max_page_data_size, page_g.max_data_size);
      }
      __syncwarp();
      if (t == 0) {
        if (not pages.empty()) pages[ck_g.first_page] = page_g;
        if (not page_sizes.empty()) page_sizes[ck_g.first_page] = page_g.max_data_size;
        if (page_grstats) page_grstats[ck_g.first_page] = pagestats_g;
      }
      num_pages = 1;
    }
    __syncwarp();
    // This loop goes over one page fragment at a time and adds it to page.
    // When page size crosses a particular limit, then it moves on to the next page and then next
    // page fragment gets added to that one.

    // This doesn't actually deal with data. It's agnostic. It only cares about number of rows and
    // page size.
    do {
      uint32_t minmax_len = 0;
      __syncwarp();
      if (num_rows < ck_g.num_rows) {
        if (t == 0) { frag_g = ck_g.fragments[fragments_in_chunk]; }
        if (!t && ck_g.stats) {
          if (col_g.stats_dtype == dtype_string) {
            minmax_len = max(ck_g.stats[fragments_in_chunk].min_value.str_val.length,
                             ck_g.stats[fragments_in_chunk].max_value.str_val.length);
          } else if (col_g.stats_dtype == dtype_byte_array) {
            minmax_len = max(ck_g.stats[fragments_in_chunk].min_value.byte_val.length,
                             ck_g.stats[fragments_in_chunk].max_value.byte_val.length);
          }
        }
      } else if (!t) {
        frag_g.fragment_data_size = 0;
        frag_g.num_rows           = 0;
      }
      __syncwarp();
      uint32_t fragment_data_size =
        (ck_g.use_dictionary)
          ? frag_g.num_leaf_values * util::div_rounding_up_unsafe(ck_g.dict_rle_bits, 8)
          : frag_g.fragment_data_size;

      // page fragment size must fit in a 32-bit signed integer
      if (fragment_data_size > std::numeric_limits<int32_t>::max()) {
        CUDF_UNREACHABLE("page fragment size exceeds maximum for i32");
      }

      // TODO (dm): this convoluted logic to limit page size needs refactoring
      size_t this_max_page_size = (values_in_page * 2 >= ck_g.num_values)   ? 256 * 1024
                                  : (values_in_page * 3 >= ck_g.num_values) ? 384 * 1024
                                                                            : 512 * 1024;

      // override this_max_page_size if the requested size is smaller
      this_max_page_size = min(this_max_page_size, max_page_size_bytes);

      // subtract size of rep and def level vectors
      auto num_vals = values_in_page + frag_g.num_values;
      this_max_page_size =
        underflow_safe_subtract(this_max_page_size,
                                max_RLE_page_size(col_g.num_def_level_bits(), num_vals) +
                                  max_RLE_page_size(col_g.num_rep_level_bits(), num_vals));

      if (num_rows >= ck_g.num_rows ||
          (values_in_page > 0 && (page_size + fragment_data_size > this_max_page_size)) ||
          rows_in_page >= max_page_size_rows) {
        if (ck_g.use_dictionary) {
          // Additional byte to store entry bit width
          page_size = 1 + max_RLE_page_size(ck_g.dict_rle_bits, values_in_page);
        }
        if (!t) {
          page_g.num_fragments = fragments_in_chunk - page_start;
          page_g.chunk         = &chunks[blockIdx.y][blockIdx.x];
          page_g.chunk_id      = blockIdx.y * num_columns + blockIdx.x;
          page_g.page_type     = PageType::DATA_PAGE;
          page_g.hdr_size      = 0;
          page_g.max_hdr_size  = 32;  // Max size excluding statistics
          if (ck_g.stats) {
            uint32_t stats_hdr_len = 16;
            if (col_g.stats_dtype == dtype_string || col_g.stats_dtype == dtype_byte_array) {
              stats_hdr_len += 5 * 3 + 2 * max_stats_len;
            } else {
              stats_hdr_len += ((col_g.stats_dtype >= dtype_int64) ? 10 : 5) * 3;
            }
            page_g.max_hdr_size += stats_hdr_len;
          }
          page_g.max_hdr_size = util::round_up_unsafe(page_g.max_hdr_size, page_align);
          page_g.page_data    = ck_g.uncompressed_bfr + page_offset;
          if (not comp_page_sizes.empty()) {
            page_g.compressed_data = ck_g.compressed_bfr + comp_page_offset;
          }
          page_g.start_row          = cur_row;
          page_g.num_rows           = rows_in_page;
          page_g.num_leaf_values    = leaf_values_in_page;
          page_g.num_values         = values_in_page;
          auto const def_level_size = max_RLE_page_size(col_g.num_def_level_bits(), values_in_page);
          auto const rep_level_size = max_RLE_page_size(col_g.num_rep_level_bits(), values_in_page);
          auto const max_data_size  = page_size + def_level_size + rep_level_size;
          // page size must fit in 32-bit signed integer
          if (max_data_size > std::numeric_limits<int32_t>::max()) {
            CUDF_UNREACHABLE("page size exceeds maximum for i32");
          }
          page_g.max_data_size    = static_cast<uint32_t>(max_data_size);
          pagestats_g.start_chunk = ck_g.first_fragment + page_start;
          pagestats_g.num_chunks  = page_g.num_fragments;
          page_offset +=
            util::round_up_unsafe(page_g.max_hdr_size + page_g.max_data_size, page_align);
          if (not comp_page_sizes.empty()) {
            comp_page_offset += page_g.max_hdr_size + comp_page_sizes[ck_g.first_page + num_pages];
          }
          page_headers_size += page_g.max_hdr_size;
          max_page_data_size = max(max_page_data_size, page_g.max_data_size);
          cur_row += rows_in_page;
          ck_max_stats_len = max(ck_max_stats_len, max_stats_len);
        }
        __syncwarp();
        if (t == 0) {
          if (not pages.empty()) { pages[ck_g.first_page + num_pages] = page_g; }
          if (not page_sizes.empty()) {
            page_sizes[ck_g.first_page + num_pages] = page_g.max_data_size;
          }
          if (page_grstats) { page_grstats[ck_g.first_page + num_pages] = pagestats_g; }
        }

        num_pages++;
        page_size           = 0;
        rows_in_page        = 0;
        values_in_page      = 0;
        leaf_values_in_page = 0;
        page_start          = fragments_in_chunk;
        max_stats_len       = 0;
      }
      max_stats_len = max(max_stats_len, minmax_len);
      num_dict_entries += frag_g.num_dict_vals;
      page_size += fragment_data_size;
      rows_in_page += frag_g.num_rows;
      values_in_page += frag_g.num_values;
      leaf_values_in_page += frag_g.num_leaf_values;
      num_rows += frag_g.num_rows;
      fragments_in_chunk++;
    } while (frag_g.num_rows != 0);
    __syncwarp();
    if (!t) {
      if (ck_g.ck_stat_size == 0 && ck_g.stats) {
        uint32_t ck_stat_size = util::round_up_unsafe(48 + 2 * ck_max_stats_len, page_align);
        page_offset += ck_stat_size;
        comp_page_offset += ck_stat_size;
        ck_g.ck_stat_size = ck_stat_size;
      }
      ck_g.num_pages          = num_pages;
      ck_g.bfr_size           = page_offset;
      ck_g.page_headers_size  = page_headers_size;
      ck_g.max_page_data_size = max_page_data_size;
      if (not comp_page_sizes.empty()) { ck_g.compressed_size = comp_page_offset; }
      pagestats_g.start_chunk = ck_g.first_page + ck_g.use_dictionary;  // Exclude dictionary
      pagestats_g.num_chunks  = num_pages - ck_g.use_dictionary;
    }
  }
  __syncthreads();
  if (t == 0) {
    if (not pages.empty()) ck_g.pages = &pages[ck_g.first_page];
    chunks[blockIdx.y][blockIdx.x] = ck_g;
    if (chunk_grstats) chunk_grstats[blockIdx.y * num_columns + blockIdx.x] = pagestats_g;
  }
}

/**
 * @brief Mask table representing how many consecutive repeats are needed to code a repeat run
 *[nbits-1]
 */
static __device__ __constant__ uint32_t kRleRunMask[24] = {
  0x00ff'ffff, 0x0fff, 0x00ff, 0x3f, 0x0f, 0x0f, 0x7, 0x7, 0x3, 0x3, 0x3, 0x3,
  0x1,         0x1,    0x1,    0x1,  0x1,  0x1,  0x1, 0x1, 0x1, 0x1, 0x1, 0x1};

/**
 * @brief Variable-length encode an integer
 */
inline __device__ uint8_t* VlqEncode(uint8_t* p, uint32_t v)
{
  while (v > 0x7f) {
    *p++ = (v | 0x80);
    v >>= 7;
  }
  *p++ = v;
  return p;
}

/**
 * @brief Pack literal values in output bitstream (1,2,3,4,5,6,8,10,12,16,20 or 24 bits per value)
 */
inline __device__ void PackLiteralsShuffle(
  uint8_t* dst, uint32_t v, uint32_t count, uint32_t w, uint32_t t)
{
  constexpr uint32_t MASK2T = 1;  // mask for 2 thread leader
  constexpr uint32_t MASK4T = 3;  // mask for 4 thread leader
  constexpr uint32_t MASK8T = 7;  // mask for 8 thread leader
  uint64_t v64;

  if (t > (count | 0x1f)) { return; }

  switch (w) {
    case 1:
      v |= shuffle_xor(v, 1) << 1;  // grab bit 1 from neighbor
      v |= shuffle_xor(v, 2) << 2;  // grab bits 2-3 from 2 lanes over
      v |= shuffle_xor(v, 4) << 4;  // grab bits 4-7 from 4 lanes over
      // sub-warp leader writes the combined bits
      if (t < count && !(t & MASK8T)) { dst[(t * w) >> 3] = v; }
      return;
    case 2:
      v |= shuffle_xor(v, 1) << 2;
      v |= shuffle_xor(v, 2) << 4;
      if (t < count && !(t & MASK4T)) { dst[(t * w) >> 3] = v; }
      return;
    case 3:
      v |= shuffle_xor(v, 1) << 3;
      v |= shuffle_xor(v, 2) << 6;
      v |= shuffle_xor(v, 4) << 12;
      if (t < count && !(t & MASK8T)) {
        dst[(t >> 3) * 3 + 0] = v;
        dst[(t >> 3) * 3 + 1] = v >> 8;
        dst[(t >> 3) * 3 + 2] = v >> 16;
      }
      return;
    case 4:
      v |= shuffle_xor(v, 1) << 4;
      if (t < count && !(t & MASK2T)) { dst[(t * w) >> 3] = v; }
      return;
    case 5:
      v |= shuffle_xor(v, 1) << 5;
      v |= shuffle_xor(v, 2) << 10;
      v64 = static_cast<uint64_t>(shuffle_xor(v, 4)) << 20 | v;
      if (t < count && !(t & MASK8T)) {
        dst[(t >> 3) * 5 + 0] = v64;
        dst[(t >> 3) * 5 + 1] = v64 >> 8;
        dst[(t >> 3) * 5 + 2] = v64 >> 16;
        dst[(t >> 3) * 5 + 3] = v64 >> 24;
        dst[(t >> 3) * 5 + 4] = v64 >> 32;
      }
      return;
    case 6:
      v |= shuffle_xor(v, 1) << 6;
      v |= shuffle_xor(v, 2) << 12;
      if (t < count && !(t & MASK4T)) {
        dst[(t >> 2) * 3 + 0] = v;
        dst[(t >> 2) * 3 + 1] = v >> 8;
        dst[(t >> 2) * 3 + 2] = v >> 16;
      }
      return;
    case 8:
      if (t < count) { dst[t] = v; }
      return;
    case 10:
      v |= shuffle_xor(v, 1) << 10;
      v64 = static_cast<uint64_t>(shuffle_xor(v, 2)) << 20 | v;
      if (t < count && !(t & MASK4T)) {
        dst[(t >> 2) * 5 + 0] = v64;
        dst[(t >> 2) * 5 + 1] = v64 >> 8;
        dst[(t >> 2) * 5 + 2] = v64 >> 16;
        dst[(t >> 2) * 5 + 3] = v64 >> 24;
        dst[(t >> 2) * 5 + 4] = v64 >> 32;
      }
      return;
    case 12:
      v |= shuffle_xor(v, 1) << 12;
      if (t < count && !(t & MASK2T)) {
        dst[(t >> 1) * 3 + 0] = v;
        dst[(t >> 1) * 3 + 1] = v >> 8;
        dst[(t >> 1) * 3 + 2] = v >> 16;
      }
      return;
    case 16:
      if (t < count) {
        dst[t * 2 + 0] = v;
        dst[t * 2 + 1] = v >> 8;
      }
      return;
    case 20:
      v64 = static_cast<uint64_t>(shuffle_xor(v, 1)) << 20 | v;
      if (t < count && !(t & MASK2T)) {
        dst[(t >> 1) * 5 + 0] = v64;
        dst[(t >> 1) * 5 + 1] = v64 >> 8;
        dst[(t >> 1) * 5 + 2] = v64 >> 16;
        dst[(t >> 1) * 5 + 3] = v64 >> 24;
        dst[(t >> 1) * 5 + 4] = v64 >> 32;
      }
      return;
    case 24:
      if (t < count) {
        dst[t * 3 + 0] = v;
        dst[t * 3 + 1] = v >> 8;
        dst[t * 3 + 2] = v >> 16;
      }
      return;

    default: CUDF_UNREACHABLE("Unsupported bit width");
  }
}

/**
 * @brief Pack literals of arbitrary bit-length in output bitstream.
 */
inline __device__ void PackLiteralsRoundRobin(
  uint8_t* dst, uint32_t v, uint32_t count, uint32_t w, uint32_t t)
{
  // Scratch space to temporarily write to. Needed because we will use atomics to write 32 bit
  // words but the destination mem may not be a multiple of 4 bytes.
  // TODO (dm): This assumes blockdim = 128. Reduce magic numbers.
  constexpr uint32_t NUM_THREADS  = 128;  // this needs to match gpuEncodePages block_size parameter
  constexpr uint32_t NUM_BYTES    = (NUM_THREADS * MAX_DICT_BITS) >> 3;
  constexpr uint32_t SCRATCH_SIZE = NUM_BYTES / sizeof(uint32_t);
  __shared__ uint32_t scratch[SCRATCH_SIZE];
  for (uint32_t i = t; i < SCRATCH_SIZE; i += NUM_THREADS) {
    scratch[i] = 0;
  }
  __syncthreads();

  if (t <= count) {
    // shift symbol left by up to 31 bits
    uint64_t v64 = v;
    v64 <<= (t * w) & 0x1f;

    // Copy 64 bit word into two 32 bit words while following C++ strict aliasing rules.
    uint32_t v32[2];
    memcpy(&v32, &v64, sizeof(uint64_t));

    // Atomically write result to scratch
    if (v32[0]) { atomicOr(scratch + ((t * w) >> 5), v32[0]); }
    if (v32[1]) { atomicOr(scratch + ((t * w) >> 5) + 1, v32[1]); }
  }
  __syncthreads();

  // Copy scratch data to final destination
  auto available_bytes = (count * w + 7) / 8;

  auto scratch_bytes = reinterpret_cast<char*>(&scratch[0]);
  for (uint32_t i = t; i < available_bytes; i += NUM_THREADS) {
    dst[i] = scratch_bytes[i];
  }
  __syncthreads();
}

/**
 * @brief Pack literal values in output bitstream
 */
inline __device__ void PackLiterals(
  uint8_t* dst, uint32_t v, uint32_t count, uint32_t w, uint32_t t)
{
  if (w > 24) { CUDF_UNREACHABLE("Unsupported bit width"); }
  switch (w) {
    case 1:
    case 2:
    case 3:
    case 4:
    case 5:
    case 6:
    case 8:
    case 10:
    case 12:
    case 16:
    case 20:
    case 24:
      // bit widths that lie on easy boundaries can be handled either directly
      // (8, 16, 24) or through fast shuffle operations.
      PackLiteralsShuffle(dst, v, count, w, t);
      return;
    default:
      // bit packing that uses atomics, but can handle arbitrary bit widths up to 24.
      PackLiteralsRoundRobin(dst, v, count, w, t);
  }
}

/**
 * @brief RLE encoder
 *
 * @param[in,out] s Page encode state
 * @param[in] numvals Total count of input values
 * @param[in] nbits number of bits per symbol (1..16)
 * @param[in] flush nonzero if last batch in block
 * @param[in] t thread id (0..127)
 */
static __device__ void RleEncode(
  page_enc_state_s* s, uint32_t numvals, uint32_t nbits, uint32_t flush, uint32_t t)
{
  uint32_t rle_pos = s->rle_pos;
  uint32_t rle_run = s->rle_run;

  while (rle_pos < numvals || (flush && rle_run)) {
    uint32_t pos = rle_pos + t;
    if (rle_run > 0 && !(rle_run & 1)) {
      // Currently in a long repeat run
      uint32_t mask = ballot(pos < numvals && s->vals[pos & (rle_buffer_size - 1)] == s->run_val);
      uint32_t rle_rpt_count, max_rpt_count;
      if (!(t & 0x1f)) { s->rpt_map[t >> 5] = mask; }
      __syncthreads();
      if (t < 32) {
        uint32_t c32 = ballot(t >= 4 || s->rpt_map[t] != 0xffff'ffffu);
        if (!t) {
          uint32_t last_idx = __ffs(c32) - 1;
          s->rle_rpt_count =
            last_idx * 32 + ((last_idx < 4) ? __ffs(~s->rpt_map[last_idx]) - 1 : 0);
        }
      }
      __syncthreads();
      max_rpt_count = min(numvals - rle_pos, 128);
      rle_rpt_count = s->rle_rpt_count;
      rle_run += rle_rpt_count << 1;
      rle_pos += rle_rpt_count;
      if (rle_rpt_count < max_rpt_count || (flush && rle_pos == numvals)) {
        if (t == 0) {
          uint32_t const run_val = s->run_val;
          uint8_t* dst           = VlqEncode(s->rle_out, rle_run);
          *dst++                 = run_val;
          if (nbits > 8) { *dst++ = run_val >> 8; }
          if (nbits > 16) { *dst++ = run_val >> 16; }
          s->rle_out = dst;
        }
        rle_run = 0;
      }
    } else {
      // New run or in a literal run
      uint32_t v0      = s->vals[pos & (rle_buffer_size - 1)];
      uint32_t v1      = s->vals[(pos + 1) & (rle_buffer_size - 1)];
      uint32_t mask    = ballot(pos + 1 < numvals && v0 == v1);
      uint32_t maxvals = min(numvals - rle_pos, 128);
      uint32_t rle_lit_count, rle_rpt_count;
      if (!(t & 0x1f)) { s->rpt_map[t >> 5] = mask; }
      __syncthreads();
      if (t < 32) {
        // Repeat run can only start on a multiple of 8 values
        uint32_t idx8        = (t * 8) >> 5;
        uint32_t pos8        = (t * 8) & 0x1f;
        uint32_t m0          = (idx8 < 4) ? s->rpt_map[idx8] : 0;
        uint32_t m1          = (idx8 < 3) ? s->rpt_map[idx8 + 1] : 0;
        uint32_t needed_mask = kRleRunMask[nbits - 1];
        mask                 = ballot((__funnelshift_r(m0, m1, pos8) & needed_mask) == needed_mask);
        if (!t) {
          uint32_t rle_run_start = (mask != 0) ? min((__ffs(mask) - 1) * 8, maxvals) : maxvals;
          uint32_t rpt_len       = 0;
          if (rle_run_start < maxvals) {
            uint32_t idx_cur = rle_run_start >> 5;
            uint32_t idx_ofs = rle_run_start & 0x1f;
            while (idx_cur < 4) {
              m0   = (idx_cur < 4) ? s->rpt_map[idx_cur] : 0;
              m1   = (idx_cur < 3) ? s->rpt_map[idx_cur + 1] : 0;
              mask = ~__funnelshift_r(m0, m1, idx_ofs);
              if (mask != 0) {
                rpt_len += __ffs(mask) - 1;
                break;
              }
              rpt_len += 32;
              idx_cur++;
            }
          }
          s->rle_lit_count = rle_run_start;
          s->rle_rpt_count = min(rpt_len, maxvals - rle_run_start);
        }
      }
      __syncthreads();
      rle_lit_count = s->rle_lit_count;
      rle_rpt_count = s->rle_rpt_count;
      if (rle_lit_count != 0 || (rle_run != 0 && rle_rpt_count != 0)) {
        uint32_t lit_div8;
        bool need_more_data = false;
        if (!flush && rle_pos + rle_lit_count == numvals) {
          // Wait for more data
          rle_lit_count -= min(rle_lit_count, 24);
          need_more_data = true;
        }
        if (rle_lit_count != 0) {
          lit_div8 = (rle_lit_count + ((flush && rle_pos + rle_lit_count == numvals) ? 7 : 0)) >> 3;
          if (rle_run + lit_div8 * 2 > 0x7f) {
            lit_div8      = 0x3f - (rle_run >> 1);  // Limit to fixed 1-byte header (504 literals)
            rle_rpt_count = 0;                      // Defer repeat run
          }
          if (lit_div8 != 0) {
            uint8_t* dst = s->rle_out + 1 + (rle_run >> 1) * nbits;
            PackLiterals(dst, (rle_pos + t < numvals) ? v0 : 0, lit_div8 * 8, nbits, t);
            rle_run = (rle_run + lit_div8 * 2) | 1;
            rle_pos = min(rle_pos + lit_div8 * 8, numvals);
          }
        }
        if (rle_run >= ((rle_rpt_count != 0 || (flush && rle_pos == numvals)) ? 0x03 : 0x7f)) {
          __syncthreads();
          // Complete literal run
          if (!t) {
            uint8_t* dst = s->rle_out;
            dst[0]       = rle_run;  // At most 0x7f
            dst += 1 + nbits * (rle_run >> 1);
            s->rle_out = dst;
          }
          rle_run = 0;
        }
        if (need_more_data) { break; }
      }
      // Start a repeat run
      if (rle_rpt_count != 0) {
        if (t == s->rle_lit_count) { s->run_val = v0; }
        rle_run = rle_rpt_count * 2;
        rle_pos += rle_rpt_count;
        if (rle_pos + 1 == numvals && !flush) { break; }
      }
    }
    __syncthreads();
  }
  __syncthreads();
  if (!t) {
    s->rle_run     = rle_run;
    s->rle_pos     = rle_pos;
    s->rle_numvals = numvals;
  }
}

/**
 * @brief PLAIN bool encoder
 *
 * @param[in,out] s Page encode state
 * @param[in] numvals Total count of input values
 * @param[in] flush nonzero if last batch in block
 * @param[in] t thread id (0..127)
 */
static __device__ void PlainBoolEncode(page_enc_state_s* s,
                                       uint32_t numvals,
                                       uint32_t flush,
                                       uint32_t t)
{
  uint32_t rle_pos = s->rle_pos;
  uint8_t* dst     = s->rle_out;

  while (rle_pos < numvals) {
    uint32_t pos    = rle_pos + t;
    uint32_t v      = (pos < numvals) ? s->vals[pos & (rle_buffer_size - 1)] : 0;
    uint32_t n      = min(numvals - rle_pos, 128);
    uint32_t nbytes = (n + ((flush) ? 7 : 0)) >> 3;
    if (!nbytes) { break; }
    v |= shuffle_xor(v, 1) << 1;
    v |= shuffle_xor(v, 2) << 2;
    v |= shuffle_xor(v, 4) << 4;
    if (t < n && !(t & 7)) { dst[t >> 3] = v; }
    rle_pos = min(rle_pos + nbytes * 8, numvals);
    dst += nbytes;
  }
  __syncthreads();
  if (!t) {
    s->rle_pos     = rle_pos;
    s->rle_numvals = numvals;
    s->rle_out     = dst;
  }
}

/**
 * @brief Determines the difference between the Proleptic Gregorian Calendar epoch (1970-01-01
 * 00:00:00 UTC) and the Julian date epoch (-4713-11-24 12:00:00 UTC).
 *
 * @return The difference between two epochs in `cuda::std::chrono::duration` format with a period
 * of hours.
 */
constexpr auto julian_calendar_epoch_diff()
{
  using namespace cuda::std::chrono;
  using namespace cuda::std::chrono_literals;
  return sys_days{January / 1 / 1970} - (sys_days{November / 24 / -4713} + 12h);
}

/**
 * @brief Converts a timestamp_ns into a pair with nanoseconds since midnight and number of Julian
 * days. Does not deal with time zones. Used by INT96 code.
 *
 * @param ns number of nanoseconds since epoch
 * @return std::pair<nanoseconds,days> where nanoseconds is the number of nanoseconds
 * elapsed in the day and days is the number of days from Julian epoch.
 */
static __device__ std::pair<duration_ns, duration_D> convert_nanoseconds(timestamp_ns const ns)
{
  using namespace cuda::std::chrono;
  auto const nanosecond_ticks = ns.time_since_epoch();
  auto const gregorian_days   = floor<days>(nanosecond_ticks);
  auto const julian_days      = gregorian_days + ceil<days>(julian_calendar_epoch_diff());

  auto const last_day_ticks = nanosecond_ticks - gregorian_days;
  return {last_day_ticks, julian_days};
}

// blockDim(128, 1, 1)
template <int block_size>
__global__ void __launch_bounds__(128, 8)
  gpuEncodePages(device_span<gpu::EncPage> pages,
                 device_span<device_span<uint8_t const>> comp_in,
                 device_span<device_span<uint8_t>> comp_out,
                 device_span<compression_result> comp_results)
{
  __shared__ __align__(8) page_enc_state_s state_g;
  using block_scan = hipcub::BlockScan<uint32_t, block_size>;
  __shared__ typename block_scan::TempStorage temp_storage;

  page_enc_state_s* const s = &state_g;
  uint32_t t                = threadIdx.x;

  if (t == 0) {
    state_g = page_enc_state_s{};
    s->page = pages[blockIdx.x];
    s->ck   = *s->page.chunk;
    s->col  = *s->ck.col_desc;
    s->cur  = s->page.page_data + s->page.max_hdr_size;
  }
  __syncthreads();

  // Encode Repetition and Definition levels
  if (s->page.page_type != PageType::DICTIONARY_PAGE &&
      (s->col.num_def_level_bits()) != 0 &&  // This means max definition level is not 0 (nullable)
      (s->col.num_rep_level_bits()) == 0     // This means there are no repetition levels (non-list)
  ) {
    // Calculate definition levels from validity
    uint32_t def_lvl_bits = s->col.num_def_level_bits();
    if (def_lvl_bits != 0) {
      if (!t) {
        s->rle_run     = 0;
        s->rle_pos     = 0;
        s->rle_numvals = 0;
        s->rle_out     = s->cur + 4;
      }
      __syncthreads();
      while (s->rle_numvals < s->page.num_rows) {
        uint32_t rle_numvals = s->rle_numvals;
        uint32_t nrows       = min(s->page.num_rows - rle_numvals, 128);
        uint32_t row         = s->page.start_row + rle_numvals + t;
        // Definition level encodes validity. Checks the valid map and if it is valid, then sets the
        // def_lvl accordingly and sets it in s->vals which is then given to RleEncode to encode
        uint32_t def_lvl = [&]() {
          bool within_bounds = rle_numvals + t < s->page.num_rows && row < s->col.num_rows;
          if (not within_bounds) { return 0u; }
          uint32_t def       = 0;
          size_type l        = 0;
          bool is_col_struct = false;
          auto col           = *s->col.parent_column;
          do {
            // If col not nullable then it does not contribute to def levels
            if (s->col.nullability[l]) {
              if (col.is_valid(row)) {
                ++def;
              } else {
                // We have found the shallowest level at which this row is null
                break;
              }
            }
            is_col_struct = (col.type().id() == type_id::STRUCT);
            if (is_col_struct) {
              row += col.offset();
              col = col.child(0);
              ++l;
            }
          } while (is_col_struct);
          return def;
        }();
        s->vals[(rle_numvals + t) & (rle_buffer_size - 1)] = def_lvl;
        __syncthreads();
        rle_numvals += nrows;
        RleEncode(s, rle_numvals, def_lvl_bits, (rle_numvals == s->page.num_rows), t);
        __syncthreads();
      }
      if (t < 32) {
        uint8_t* cur     = s->cur;
        uint8_t* rle_out = s->rle_out;
        if (t < 4) {
          uint32_t rle_bytes = (uint32_t)(rle_out - cur) - 4;
          cur[t]             = rle_bytes >> (t * 8);
        }
        __syncwarp();
        if (t == 0) { s->cur = rle_out; }
      }
    }
  } else if (s->page.page_type != PageType::DICTIONARY_PAGE &&
             s->col.num_rep_level_bits() != 0  // This means there ARE repetition levels (has list)
  ) {
    auto encode_levels = [&](uint8_t const* lvl_val_data, uint32_t nbits) {
      // For list types, the repetition and definition levels are pre-calculated. We just need to
      // encode and write them now.
      if (!t) {
        s->rle_run     = 0;
        s->rle_pos     = 0;
        s->rle_numvals = 0;
        s->rle_out     = s->cur + 4;
      }
      __syncthreads();
      size_type page_first_val_idx = s->col.level_offsets[s->page.start_row];
      size_type col_last_val_idx   = s->col.level_offsets[s->col.num_rows];
      while (s->rle_numvals < s->page.num_values) {
        uint32_t rle_numvals = s->rle_numvals;
        uint32_t nvals       = min(s->page.num_values - rle_numvals, 128);
        uint32_t idx         = page_first_val_idx + rle_numvals + t;
        uint32_t lvl_val =
          (rle_numvals + t < s->page.num_values && idx < col_last_val_idx) ? lvl_val_data[idx] : 0;
        s->vals[(rle_numvals + t) & (rle_buffer_size - 1)] = lvl_val;
        __syncthreads();
        rle_numvals += nvals;
        RleEncode(s, rle_numvals, nbits, (rle_numvals == s->page.num_values), t);
        __syncthreads();
      }
      if (t < 32) {
        uint8_t* cur     = s->cur;
        uint8_t* rle_out = s->rle_out;
        if (t < 4) {
          uint32_t rle_bytes = (uint32_t)(rle_out - cur) - 4;
          cur[t]             = rle_bytes >> (t * 8);
        }
        __syncwarp();
        if (t == 0) { s->cur = rle_out; }
      }
    };
    encode_levels(s->col.rep_values, s->col.num_rep_level_bits());
    __syncthreads();
    encode_levels(s->col.def_values, s->col.num_def_level_bits());
  }
  // Encode data values
  __syncthreads();
  auto const physical_type = s->col.physical_type;
  auto const type_id       = s->col.leaf_column->type().id();
  auto const dtype_len_out = physical_type_len(physical_type, type_id);
  auto const dtype_len_in  = [&]() -> uint32_t {
    if (physical_type == INT32) { return int32_logical_len(type_id); }
    if (physical_type == INT96) { return sizeof(int64_t); }
    return dtype_len_out;
  }();

  auto const dict_bits = (physical_type == BOOLEAN) ? 1
                         : (s->ck.use_dictionary and s->page.page_type != PageType::DICTIONARY_PAGE)
                           ? s->ck.dict_rle_bits
                           : -1;
  if (t == 0) {
    uint8_t* dst   = s->cur;
    s->rle_run     = 0;
    s->rle_pos     = 0;
    s->rle_numvals = 0;
    s->rle_out     = dst;
    if (dict_bits >= 0 && physical_type != BOOLEAN) {
      dst[0]     = dict_bits;
      s->rle_out = dst + 1;
    }
    s->page_start_val  = row_to_value_idx(s->page.start_row, s->col);
    s->chunk_start_val = row_to_value_idx(s->ck.start_row, s->col);
  }
  __syncthreads();
  for (uint32_t cur_val_idx = 0; cur_val_idx < s->page.num_leaf_values;) {
    uint32_t nvals = min(s->page.num_leaf_values - cur_val_idx, 128);
    uint32_t len, pos;

    auto [is_valid, val_idx] = [&]() {
      uint32_t val_idx;
      uint32_t is_valid;

      size_type val_idx_in_block = cur_val_idx + t;
      if (s->page.page_type == PageType::DICTIONARY_PAGE) {
        val_idx  = val_idx_in_block;
        is_valid = (val_idx < s->page.num_leaf_values);
        if (is_valid) { val_idx = s->ck.dict_data[val_idx]; }
      } else {
        size_type val_idx_in_leaf_col = s->page_start_val + val_idx_in_block;

        is_valid = (val_idx_in_leaf_col < s->col.leaf_column->size() &&
                    val_idx_in_block < s->page.num_leaf_values)
                     ? s->col.leaf_column->is_valid(val_idx_in_leaf_col)
                     : 0;
        val_idx =
          (s->ck.use_dictionary) ? val_idx_in_leaf_col - s->chunk_start_val : val_idx_in_leaf_col;
      }
      return std::make_tuple(is_valid, val_idx);
    }();

    cur_val_idx += nvals;
    if (dict_bits >= 0) {
      // Dictionary encoding
      if (dict_bits > 0) {
        uint32_t rle_numvals;
        uint32_t rle_numvals_in_block;
        block_scan(temp_storage).ExclusiveSum(is_valid, pos, rle_numvals_in_block);
        rle_numvals = s->rle_numvals;
        if (is_valid) {
          uint32_t v;
          if (physical_type == BOOLEAN) {
            v = s->col.leaf_column->element<uint8_t>(val_idx);
          } else {
            v = s->ck.dict_index[val_idx];
          }
          s->vals[(rle_numvals + pos) & (rle_buffer_size - 1)] = v;
        }
        rle_numvals += rle_numvals_in_block;
        __syncthreads();
        if ((!enable_bool_rle) && (physical_type == BOOLEAN)) {
          PlainBoolEncode(s, rle_numvals, (cur_val_idx == s->page.num_leaf_values), t);
        } else {
          RleEncode(s, rle_numvals, dict_bits, (cur_val_idx == s->page.num_leaf_values), t);
        }
        __syncthreads();
      }
      if (t == 0) { s->cur = s->rle_out; }
      __syncthreads();
    } else {
      // Non-dictionary encoding
      uint8_t* dst = s->cur;

      if (is_valid) {
        len = dtype_len_out;
        if (physical_type == BYTE_ARRAY) {
          if (type_id == type_id::STRING) {
            len += s->col.leaf_column->element<string_view>(val_idx).size_bytes();
          } else if (s->col.output_as_byte_array && type_id == type_id::LIST) {
            len +=
              get_element<statistics::byte_array_view>(*s->col.leaf_column, val_idx).size_bytes();
          }
        }
      } else {
        len = 0;
      }
      uint32_t total_len = 0;
      block_scan(temp_storage).ExclusiveSum(len, pos, total_len);
      __syncthreads();
      if (t == 0) { s->cur = dst + total_len; }
      if (is_valid) {
        switch (physical_type) {
          case INT32: [[fallthrough]];
          case FLOAT: {
            auto const v = [dtype_len = dtype_len_in,
                            idx       = val_idx,
                            col       = s->col.leaf_column,
                            scale     = s->col.ts_scale == 0 ? 1 : s->col.ts_scale]() -> int32_t {
              switch (dtype_len) {
                case 8: return col->element<int64_t>(idx) * scale;
                case 4: return col->element<int32_t>(idx) * scale;
                case 2: return col->element<int16_t>(idx) * scale;
                default: return col->element<int8_t>(idx) * scale;
              }
            }();

            dst[pos + 0] = v;
            dst[pos + 1] = v >> 8;
            dst[pos + 2] = v >> 16;
            dst[pos + 3] = v >> 24;
          } break;
          case INT64: {
            int64_t v        = s->col.leaf_column->element<int64_t>(val_idx);
            int32_t ts_scale = s->col.ts_scale;
            if (ts_scale != 0) {
              if (ts_scale < 0) {
                v /= -ts_scale;
              } else {
                v *= ts_scale;
              }
            }
            dst[pos + 0] = v;
            dst[pos + 1] = v >> 8;
            dst[pos + 2] = v >> 16;
            dst[pos + 3] = v >> 24;
            dst[pos + 4] = v >> 32;
            dst[pos + 5] = v >> 40;
            dst[pos + 6] = v >> 48;
            dst[pos + 7] = v >> 56;
          } break;
          case INT96: {
            int64_t v        = s->col.leaf_column->element<int64_t>(val_idx);
            int32_t ts_scale = s->col.ts_scale;
            if (ts_scale != 0) {
              if (ts_scale < 0) {
                v /= -ts_scale;
              } else {
                v *= ts_scale;
              }
            }

            auto const ret = convert_nanoseconds([&]() {
              switch (s->col.leaf_column->type().id()) {
                case type_id::TIMESTAMP_SECONDS:
                case type_id::TIMESTAMP_MILLISECONDS: {
                  return timestamp_ns{duration_ms{v}};
                } break;
                case type_id::TIMESTAMP_MICROSECONDS:
                case type_id::TIMESTAMP_NANOSECONDS: {
                  return timestamp_ns{duration_us{v}};
                } break;
              }
              return timestamp_ns{duration_ns{0}};
            }());

            // the 12 bytes of fixed length data.
            v             = ret.first.count();
            dst[pos + 0]  = v;
            dst[pos + 1]  = v >> 8;
            dst[pos + 2]  = v >> 16;
            dst[pos + 3]  = v >> 24;
            dst[pos + 4]  = v >> 32;
            dst[pos + 5]  = v >> 40;
            dst[pos + 6]  = v >> 48;
            dst[pos + 7]  = v >> 56;
            uint32_t w    = ret.second.count();
            dst[pos + 8]  = w;
            dst[pos + 9]  = w >> 8;
            dst[pos + 10] = w >> 16;
            dst[pos + 11] = w >> 24;
          } break;

          case DOUBLE: {
            auto v = s->col.leaf_column->element<double>(val_idx);
            memcpy(dst + pos, &v, 8);
          } break;
          case BYTE_ARRAY: {
            auto const bytes = [](cudf::type_id const type_id,
                                  column_device_view const* leaf_column,
                                  uint32_t const val_idx) -> void const* {
              switch (type_id) {
                case type_id::STRING:
                  return reinterpret_cast<void const*>(
                    leaf_column->element<string_view>(val_idx).data());
                case type_id::LIST:
                  return reinterpret_cast<void const*>(
                    get_element<statistics::byte_array_view>(*(leaf_column), val_idx).data());
                default: CUDF_UNREACHABLE("invalid type id for byte array writing!");
              }
            }(type_id, s->col.leaf_column, val_idx);
            uint32_t v   = len - 4;  // string length
            dst[pos + 0] = v;
            dst[pos + 1] = v >> 8;
            dst[pos + 2] = v >> 16;
            dst[pos + 3] = v >> 24;
            if (v != 0) memcpy(dst + pos + 4, bytes, v);
          } break;
          case FIXED_LEN_BYTE_ARRAY: {
            if (type_id == type_id::DECIMAL128) {
              // When using FIXED_LEN_BYTE_ARRAY for decimals, the rep is encoded in big-endian
              auto const v = s->col.leaf_column->element<numeric::decimal128>(val_idx).value();
              auto const v_char_ptr = reinterpret_cast<char const*>(&v);
              thrust::copy(thrust::seq,
                           thrust::make_reverse_iterator(v_char_ptr + sizeof(v)),
                           thrust::make_reverse_iterator(v_char_ptr),
                           dst + pos);
            }
          } break;
        }
      }
      __syncthreads();
    }
  }
  if (t == 0) {
    uint8_t* base         = s->page.page_data + s->page.max_hdr_size;
    auto actual_data_size = static_cast<uint32_t>(s->cur - base);
    s->page.max_data_size = actual_data_size;
    if (not comp_in.empty()) {
      comp_in[blockIdx.x]  = {base, actual_data_size};
      comp_out[blockIdx.x] = {s->page.compressed_data + s->page.max_hdr_size, 0};  // size is unused
    }
    pages[blockIdx.x] = s->page;
    if (not comp_results.empty()) {
      comp_results[blockIdx.x]   = {0, compression_status::FAILURE};
      pages[blockIdx.x].comp_res = &comp_results[blockIdx.x];
    }
  }
}

// blockDim(128, 1, 1)
__global__ void __launch_bounds__(128) gpuDecideCompression(device_span<EncColumnChunk> chunks)
{
  // After changing the way structs are loaded from coop to normal, this kernel has no business
  // being launched with 128 thread block. It can easily be a single warp.
  __shared__ __align__(8) EncColumnChunk ck_g;
  __shared__ __align__(4) unsigned int error_count;
  using warp_reduce = hipcub::WarpReduce<uint32_t>;
  __shared__ typename warp_reduce::TempStorage temp_storage[2];
  __shared__ volatile bool has_compression;

  uint32_t t                      = threadIdx.x;
  uint32_t uncompressed_data_size = 0;
  uint32_t compressed_data_size   = 0;
  uint32_t num_pages;

  if (t == 0) {
    ck_g = chunks[blockIdx.x];
    atomicAnd(&error_count, 0);
    has_compression = false;
  }
  __syncthreads();
  if (t < 32) {
    num_pages = ck_g.num_pages;
    for (uint32_t page = t; page < num_pages; page += 32) {
      auto& curr_page         = ck_g.pages[page];
      uint32_t page_data_size = curr_page.max_data_size;
      uncompressed_data_size += page_data_size;
      if (auto comp_res = curr_page.comp_res; comp_res != nullptr) {
        has_compression = true;
        compressed_data_size += comp_res->bytes_written;
        if (comp_res->status != compression_status::SUCCESS) { atomicAdd(&error_count, 1); }
      }
    }
    uncompressed_data_size = warp_reduce(temp_storage[0]).Sum(uncompressed_data_size);
    compressed_data_size   = warp_reduce(temp_storage[1]).Sum(compressed_data_size);
  }
  __syncthreads();
  if (t == 0) {
    bool is_compressed;
    if (has_compression) {
      uint32_t compression_error = atomicAdd(&error_count, 0);
      is_compressed = (!compression_error && compressed_data_size < uncompressed_data_size);
    } else {
      is_compressed = false;
    }
    chunks[blockIdx.x].is_compressed = is_compressed;
    chunks[blockIdx.x].bfr_size      = uncompressed_data_size;
    chunks[blockIdx.x].compressed_size =
      (is_compressed) ? compressed_data_size : uncompressed_data_size;
  }
}

/**
 * Minimal thrift compact protocol support
 */
inline __device__ uint8_t* cpw_put_uint8(uint8_t* p, uint8_t v)
{
  *p++ = v;
  return p;
}

inline __device__ uint8_t* cpw_put_uint32(uint8_t* p, uint32_t v)
{
  while (v > 0x7f) {
    *p++ = v | 0x80;
    v >>= 7;
  }
  *p++ = v;
  return p;
}

inline __device__ uint8_t* cpw_put_uint64(uint8_t* p, uint64_t v)
{
  while (v > 0x7f) {
    *p++ = v | 0x80;
    v >>= 7;
  }
  *p++ = v;
  return p;
}

inline __device__ uint8_t* cpw_put_int32(uint8_t* p, int32_t v)
{
  int32_t s = (v < 0);
  return cpw_put_uint32(p, (v ^ -s) * 2 + s);
}

inline __device__ uint8_t* cpw_put_int64(uint8_t* p, int64_t v)
{
  int64_t s = (v < 0);
  return cpw_put_uint64(p, (v ^ -s) * 2 + s);
}

inline __device__ uint8_t* cpw_put_fldh(uint8_t* p, int f, int cur, int t)
{
  if (f > cur && f <= cur + 15) {
    *p++ = ((f - cur) << 4) | t;
    return p;
  } else {
    *p++ = t;
    return cpw_put_int32(p, f);
  }
}

class header_encoder {
  uint8_t* current_header_ptr;
  int current_field_index;

 public:
  inline __device__ header_encoder(uint8_t* header_start)
    : current_header_ptr(header_start), current_field_index(0)
  {
  }

  inline __device__ void field_struct_begin(int field)
  {
    current_header_ptr =
      cpw_put_fldh(current_header_ptr, field, current_field_index, ST_FLD_STRUCT);
    current_field_index = 0;
  }

  inline __device__ void field_struct_end(int field)
  {
    *current_header_ptr++ = 0;
    current_field_index   = field;
  }

  inline __device__ void field_list_begin(int field, size_t len, int type)
  {
    current_header_ptr = cpw_put_fldh(current_header_ptr, field, current_field_index, ST_FLD_LIST);
    current_header_ptr = cpw_put_uint8(
      current_header_ptr, static_cast<uint8_t>((std::min(len, size_t{0xfu}) << 4) | type));
    if (len >= 0xf) { current_header_ptr = cpw_put_uint32(current_header_ptr, len); }
    current_field_index = 0;
  }

  inline __device__ void field_list_end(int field) { current_field_index = field; }

  inline __device__ void put_bool(bool value)
  {
    current_header_ptr = cpw_put_uint8(current_header_ptr, value ? ST_FLD_TRUE : ST_FLD_FALSE);
  }

  inline __device__ void put_binary(const void* value, uint32_t length)
  {
    current_header_ptr = cpw_put_uint32(current_header_ptr, length);
    memcpy(current_header_ptr, value, length);
    current_header_ptr += length;
  }

  template <typename T>
  inline __device__ void put_int64(T value)
  {
    current_header_ptr = cpw_put_int64(current_header_ptr, static_cast<int64_t>(value));
  }

  template <typename T>
  inline __device__ void field_int32(int field, T value)
  {
    current_header_ptr  = cpw_put_fldh(current_header_ptr, field, current_field_index, ST_FLD_I32);
    current_header_ptr  = cpw_put_int32(current_header_ptr, static_cast<int32_t>(value));
    current_field_index = field;
  }

  template <typename T>
  inline __device__ void field_int64(int field, T value)
  {
    current_header_ptr  = cpw_put_fldh(current_header_ptr, field, current_field_index, ST_FLD_I64);
    current_header_ptr  = cpw_put_int64(current_header_ptr, static_cast<int64_t>(value));
    current_field_index = field;
  }

  inline __device__ void field_binary(int field, const void* value, uint32_t length)
  {
    current_header_ptr =
      cpw_put_fldh(current_header_ptr, field, current_field_index, ST_FLD_BINARY);
    current_header_ptr = cpw_put_uint32(current_header_ptr, length);
    memcpy(current_header_ptr, value, length);
    current_header_ptr += length;
    current_field_index = field;
  }

  inline __device__ void end(uint8_t** header_end, bool termination_flag = true)
  {
    if (not termination_flag) { *current_header_ptr++ = 0; }
    *header_end = current_header_ptr;
  }

  inline __device__ uint8_t* get_ptr() { return current_header_ptr; }

  inline __device__ void set_ptr(uint8_t* ptr) { current_header_ptr = ptr; }
};

namespace {

// byteswap 128 bit integer, placing result in dst in network byte order.
// dst must point to at least 16 bytes of memory.
__device__ void byte_reverse128(__int128_t v, void* dst)
{
  auto const v_char_ptr = reinterpret_cast<unsigned char const*>(&v);
  auto const d_char_ptr = static_cast<unsigned char*>(dst);
  thrust::copy(thrust::seq,
               thrust::make_reverse_iterator(v_char_ptr + sizeof(v)),
               thrust::make_reverse_iterator(v_char_ptr),
               d_char_ptr);
}

/**
 * @brief Test to see if a span contains all valid UTF-8 characters.
 *
 * @param span device_span to test.
 * @return true if the span contains all valid UTF-8 characters.
 */
__device__ bool is_valid_utf8(device_span<unsigned char const> span)
{
  auto idx = 0;
  while (idx < span.size_bytes()) {
    // UTF-8 character should start with valid beginning bit pattern
    if (not strings::detail::is_valid_begin_utf8_char(span[idx])) { return false; }
    // subsequent elements of the character should be continuation chars
    auto const width = strings::detail::bytes_in_utf8_byte(span[idx++]);
    for (size_type i = 1; i < width && idx < span.size_bytes(); i++, idx++) {
      if (not strings::detail::is_utf8_continuation_char(span[idx])) { return false; }
    }
  }

  return true;
}

/**
 * @brief Increment part of a UTF-8 character.
 *
 * Attempt to increment the char pointed to by ptr, which is assumed to be part of a valid UTF-8
 * character. Returns true if successful, false if the increment caused an overflow, in which case
 * the data at ptr will be set to the lowest valid UTF-8 bit pattern (start or continuation).
 * Will halt execution if passed invalid UTF-8.
 */
__device__ bool increment_utf8_at(unsigned char* ptr)
{
  unsigned char elem = *ptr;
  // elem is one of (no 5 or 6 byte chars allowed):
  //  0b0vvvvvvv a 1 byte character
  //  0b10vvvvvv a continuation byte
  //  0b110vvvvv start of a 2 byte character
  //  0b1110vvvv start of a 3 byte character
  //  0b11110vvv start of a 4 byte character

  // TODO(ets): starting at 4 byte and working down.  Should probably start low and work higher.
  uint8_t mask  = 0xF8;
  uint8_t valid = 0xF0;

  while (mask != 0) {
    if ((elem & mask) == valid) {
      elem++;
      if ((elem & mask) != mask) {  // no overflow
        *ptr = elem;
        return true;
      }
      *ptr = valid;
      return false;
    }
    mask <<= 1;
    valid <<= 1;
  }

  // should not reach here since we test for valid UTF-8 higher up the call chain
  CUDF_UNREACHABLE("Trying to increment non-utf8");
}

/**
 * @brief Attempt to truncate a span of UTF-8 characters to at most truncate_length_bytes.
 *
 * If is_min is false, then the final character (or characters if there is overflow) will be
 * incremented so that the resultant UTF-8 will still be a valid maximum. scratch is only used when
 * is_min is false, and must be at least truncate_length bytes in size. If the span cannot be
 * truncated, leave it untouched and return the original length.
 *
 * @return Pair object containing a pointer to the truncated data and its length.
 */
__device__ std::pair<const void*, uint32_t> truncate_utf8(device_span<unsigned char const> span,
                                                          bool is_min,
                                                          void* scratch,
                                                          int32_t truncate_length)
{
  // we know at this point that truncate_length < size_bytes, so
  // there is data at [len]. work backwards until we find
  // the start of a UTF-8 encoded character, since UTF-8 characters may be multi-byte.
  auto len = truncate_length;
  while (not strings::detail::is_begin_utf8_char(span[len]) && len > 0) {
    len--;
  }

  if (len != 0) {
    if (is_min) { return {span.data(), len}; }
    memcpy(scratch, span.data(), len);
    // increment last byte, working backwards if the byte overflows
    auto const ptr = static_cast<unsigned char*>(scratch);
    for (int32_t i = len - 1; i >= 0; i--) {
      if (increment_utf8_at(&ptr[i])) {  // true if no overflow
        return {scratch, len};
      }
    }
    // cannot increment, so fall through
  }

  // couldn't truncate, return original value
  return {span.data(), span.size_bytes()};
}

/**
 * @brief Attempt to truncate a span of binary data to at most truncate_length bytes.
 *
 * If is_min is false, then the final byte (or bytes if there is overflow) will be
 * incremented so that the resultant binary will still be a valid maximum. scratch is only used when
 * is_min is false, and must be at least truncate_length bytes in size. If the span cannot be
 * truncated, leave it untouched and return the original length.
 *
 * @return Pair object containing a pointer to the truncated data and its length.
 */
__device__ std::pair<const void*, uint32_t> truncate_binary(device_span<uint8_t const> arr,
                                                            bool is_min,
                                                            void* scratch,
                                                            int32_t truncate_length)
{
  if (is_min) { return {arr.data(), truncate_length}; }
  memcpy(scratch, arr.data(), truncate_length);
  // increment last byte, working backwards if the byte overflows
  auto const ptr = static_cast<uint8_t*>(scratch);
  for (int32_t i = truncate_length - 1; i >= 0; i--) {
    ptr[i]++;
    if (ptr[i] != 0) {  // no overflow
      return {scratch, i + 1};
    }
  }

  // couldn't truncate, return original value
  return {arr.data(), arr.size_bytes()};
}

// TODO (ets): the assumption here is that string columns might have UTF-8 or plain binary,
// while binary columns are assumed to be binary and will be treated as such.  If this assumption
// is incorrect, then truncate_byte_array() and truncate_string() should just be combined into
// a single function.
/**
 * @brief Attempt to truncate a UTF-8 string to at most truncate_length bytes.
 */
__device__ std::pair<const void*, uint32_t> truncate_string(const string_view& str,
                                                            bool is_min,
                                                            void* scratch,
                                                            int32_t truncate_length)
{
  if (truncate_length == NO_TRUNC_STATS or str.size_bytes() <= truncate_length) {
    return {str.data(), str.size_bytes()};
  }

  // convert char to unsigned since UTF-8 is just bytes, not chars.  can't use std::byte because
  // that can't be incremented.
  auto const span = device_span<unsigned char const>(
    reinterpret_cast<unsigned char const*>(str.data()), str.size_bytes());

  // if str is all 8-bit chars, or is actually not UTF-8, then we can just use truncate_binary()
  if (str.size_bytes() != str.length() and is_valid_utf8(span.first(truncate_length))) {
    return truncate_utf8(span, is_min, scratch, truncate_length);
  }
  return truncate_binary(span, is_min, scratch, truncate_length);
}

/**
 * @brief Attempt to truncate a binary array to at most truncate_length bytes.
 */
__device__ std::pair<const void*, uint32_t> truncate_byte_array(
  const statistics::byte_array_view& arr, bool is_min, void* scratch, int32_t truncate_length)
{
  if (truncate_length == NO_TRUNC_STATS or arr.size_bytes() <= truncate_length) {
    return {arr.data(), arr.size_bytes()};
  }

  // convert std::byte to uint8_t since bytes can't be incremented
  device_span<uint8_t const> const span{reinterpret_cast<uint8_t const*>(arr.data()),
                                        arr.size_bytes()};
  return truncate_binary(span, is_min, scratch, truncate_length);
}

/**
 * @brief Find a min or max value of the proper form to be included in Parquet statistics
 * structures.
 *
 * Given a statistics_val union and a data type, perform any transformations needed to produce a
 * valid min or max binary value.  String and byte array types will be truncated if they exceed
 * truncate_length.
 */
__device__ std::pair<const void*, uint32_t> get_extremum(const statistics_val* stats_val,
                                                         statistics_dtype dtype,
                                                         void* scratch,
                                                         bool is_min,
                                                         int32_t truncate_length)
{
  switch (dtype) {
    case dtype_bool: return {stats_val, sizeof(bool)};
    case dtype_int8:
    case dtype_int16:
    case dtype_int32:
    case dtype_date32: return {stats_val, sizeof(int32_t)};
    case dtype_float32: {
      auto const fp_scratch = static_cast<float*>(scratch);
      fp_scratch[0]         = stats_val->fp_val;
      return {scratch, sizeof(float)};
    }
    case dtype_int64:
    case dtype_timestamp64:
    case dtype_float64:
    case dtype_decimal64: return {stats_val, sizeof(int64_t)};
    case dtype_decimal128:
      byte_reverse128(stats_val->d128_val, scratch);
      return {scratch, sizeof(__int128_t)};
    case dtype_string: return truncate_string(stats_val->str_val, is_min, scratch, truncate_length);
    case dtype_byte_array:
      return truncate_byte_array(stats_val->byte_val, is_min, scratch, truncate_length);
    default: CUDF_UNREACHABLE("Invalid statistics data type");
  }
}

}  // namespace

__device__ uint8_t* EncodeStatistics(uint8_t* start,
                                     const statistics_chunk* s,
                                     statistics_dtype dtype,
                                     void* scratch)
{
  uint8_t* end;
  header_encoder encoder(start);
  encoder.field_int64(3, s->null_count);
  if (s->has_minmax) {
    auto const [max_ptr, max_size] =
      get_extremum(&s->max_value, dtype, scratch, false, NO_TRUNC_STATS);
    encoder.field_binary(5, max_ptr, max_size);
    auto const [min_ptr, min_size] =
      get_extremum(&s->min_value, dtype, scratch, true, NO_TRUNC_STATS);
    encoder.field_binary(6, min_ptr, min_size);
  }
  encoder.end(&end);
  return end;
}

// blockDim(128, 1, 1)
__global__ void __launch_bounds__(128)
  gpuEncodePageHeaders(device_span<EncPage> pages,
                       device_span<compression_result const> comp_results,
                       device_span<statistics_chunk const> page_stats,
                       const statistics_chunk* chunk_stats)
{
  // When this whole kernel becomes single thread, the following variables need not be __shared__
  __shared__ __align__(8) parquet_column_device_view col_g;
  __shared__ __align__(8) EncColumnChunk ck_g;
  __shared__ __align__(8) EncPage page_g;
  __shared__ __align__(8) unsigned char scratch[MIN_STATS_SCRATCH_SIZE];

  uint32_t t = threadIdx.x;

  if (t == 0) {
    uint8_t *hdr_start, *hdr_end;
    uint32_t compressed_page_size, uncompressed_page_size;

    page_g = pages[blockIdx.x];
    ck_g   = *page_g.chunk;
    col_g  = *ck_g.col_desc;

    if (chunk_stats && &pages[blockIdx.x] == ck_g.pages) {  // Is this the first page in a chunk?
      hdr_start = (ck_g.is_compressed) ? ck_g.compressed_bfr : ck_g.uncompressed_bfr;
      hdr_end =
        EncodeStatistics(hdr_start, &chunk_stats[page_g.chunk_id], col_g.stats_dtype, scratch);
      page_g.chunk->ck_stat_size = static_cast<uint32_t>(hdr_end - hdr_start);
    }
    uncompressed_page_size = page_g.max_data_size;
    if (ck_g.is_compressed) {
      hdr_start            = page_g.compressed_data;
      compressed_page_size = (uint32_t)comp_results[blockIdx.x].bytes_written;
      page_g.max_data_size = compressed_page_size;
    } else {
      hdr_start            = page_g.page_data;
      compressed_page_size = uncompressed_page_size;
    }
    header_encoder encoder(hdr_start);
    PageType page_type = page_g.page_type;
    // NOTE: For dictionary encoding, parquet v2 recommends using PLAIN in dictionary page and
    // RLE_DICTIONARY in data page, but parquet v1 uses PLAIN_DICTIONARY in both dictionary and
    // data pages (actual encoding is identical).
    Encoding encoding;
    if (enable_bool_rle) {
      encoding = (col_g.physical_type == BOOLEAN) ? Encoding::RLE
                 : (page_type == PageType::DICTIONARY_PAGE || page_g.chunk->use_dictionary)
                   ? Encoding::PLAIN_DICTIONARY
                   : Encoding::PLAIN;
    } else {
      encoding = (page_type == PageType::DICTIONARY_PAGE || page_g.chunk->use_dictionary)
                   ? Encoding::PLAIN_DICTIONARY
                   : Encoding::PLAIN;
    }
    encoder.field_int32(1, page_type);
    encoder.field_int32(2, uncompressed_page_size);
    encoder.field_int32(3, compressed_page_size);
    if (page_type == PageType::DATA_PAGE) {
      // DataPageHeader
      encoder.field_struct_begin(5);
      encoder.field_int32(1, page_g.num_values);  // NOTE: num_values != num_rows for list types
      encoder.field_int32(2, encoding);           // encoding
      encoder.field_int32(3, Encoding::RLE);      // definition_level_encoding
      encoder.field_int32(4, Encoding::RLE);      // repetition_level_encoding
      // Optionally encode page-level statistics
      if (not page_stats.empty()) {
        encoder.field_struct_begin(5);
        encoder.set_ptr(
          EncodeStatistics(encoder.get_ptr(), &page_stats[blockIdx.x], col_g.stats_dtype, scratch));
        encoder.field_struct_end(5);
      }
      encoder.field_struct_end(5);
    } else {
      // DictionaryPageHeader
      encoder.field_struct_begin(7);
      encoder.field_int32(1, ck_g.num_dict_entries);  // number of values in dictionary
      encoder.field_int32(2, encoding);
      encoder.field_struct_end(7);
    }
    encoder.end(&hdr_end, false);
    page_g.hdr_size = (uint32_t)(hdr_end - hdr_start);
  }
  __syncthreads();
  if (t == 0) pages[blockIdx.x] = page_g;
}

// blockDim(1024, 1, 1)
__global__ void __launch_bounds__(1024)
  gpuGatherPages(device_span<EncColumnChunk> chunks, device_span<gpu::EncPage const> pages)
{
  __shared__ __align__(8) EncColumnChunk ck_g;
  __shared__ __align__(8) EncPage page_g;

  uint32_t t = threadIdx.x;
  uint8_t *dst, *dst_base;
  const EncPage* first_page;
  uint32_t num_pages, uncompressed_size;

  if (t == 0) ck_g = chunks[blockIdx.x];
  __syncthreads();

  first_page = ck_g.pages;
  num_pages  = ck_g.num_pages;
  dst        = (ck_g.is_compressed) ? ck_g.compressed_bfr : ck_g.uncompressed_bfr;
  dst += ck_g.ck_stat_size;  // Skip over chunk statistics
  dst_base          = dst;
  uncompressed_size = ck_g.bfr_size;
  for (uint32_t page = 0; page < num_pages; page++) {
    const uint8_t* src;
    uint32_t hdr_len, data_len;

    if (t == 0) { page_g = first_page[page]; }
    __syncthreads();

    src = (ck_g.is_compressed) ? page_g.compressed_data : page_g.page_data;
    // Copy page header
    hdr_len = page_g.hdr_size;
    memcpy_block<1024, true>(dst, src, hdr_len, t);
    src += page_g.max_hdr_size;
    dst += hdr_len;
    // Copy page data
    uncompressed_size += hdr_len;
    data_len = page_g.max_data_size;
    memcpy_block<1024, true>(dst, src, data_len, t);
    dst += data_len;
    __syncthreads();
    if (!t && page == 0 && ck_g.use_dictionary) { ck_g.dictionary_size = hdr_len + data_len; }
  }
  if (t == 0) {
    chunks[blockIdx.x].bfr_size        = uncompressed_size;
    chunks[blockIdx.x].compressed_size = (dst - dst_base);
    if (ck_g.use_dictionary) { chunks[blockIdx.x].dictionary_size = ck_g.dictionary_size; }
  }
}

namespace {

/**
 * @brief Tests if statistics are comparable given the column's
 * physical and converted types
 */
__device__ bool is_comparable(Type ptype, ConvertedType ctype)
{
  switch (ptype) {
    case Type::BOOLEAN:
    case Type::INT32:
    case Type::INT64:
    case Type::FLOAT:
    case Type::DOUBLE:
    case Type::BYTE_ARRAY: return true;
    case Type::FIXED_LEN_BYTE_ARRAY:
      if (ctype == ConvertedType::DECIMAL) { return true; }
      [[fallthrough]];
    default: return false;
  }
}

/**
 * @brief Compares two values.
 * @return -1 if v1 < v2, 0 if v1 == v2, 1 if v1 > v2
 */
template <typename T>
constexpr __device__ int32_t compare(T& v1, T& v2)
{
  return (v1 > v2) - (v1 < v2);
}

/**
 * @brief Compares two statistics_val structs.
 * @return < 0 if v1 < v2, 0 if v1 == v2, > 0 if v1 > v2
 */
__device__ int32_t compare_values(Type ptype,
                                  ConvertedType ctype,
                                  const statistics_val& v1,
                                  const statistics_val& v2)
{
  switch (ptype) {
    case Type::BOOLEAN: return compare(v1.u_val, v2.u_val);
    case Type::INT32:
    case Type::INT64:
      switch (ctype) {
        case ConvertedType::UINT_8:
        case ConvertedType::UINT_16:
        case ConvertedType::UINT_32:
        case ConvertedType::UINT_64: return compare(v1.u_val, v2.u_val);
        default:  // assume everything else is signed
          return compare(v1.i_val, v2.i_val);
      }
    case Type::FLOAT:
    case Type::DOUBLE: return compare(v1.fp_val, v2.fp_val);
    case Type::BYTE_ARRAY: return static_cast<string_view>(v1.str_val).compare(v2.str_val);
    case Type::FIXED_LEN_BYTE_ARRAY:
      if (ctype == ConvertedType::DECIMAL) { return compare(v1.d128_val, v2.d128_val); }
  }
  // calling is_comparable() should prevent reaching here
  CUDF_UNREACHABLE("Trying to compare non-comparable type");
  return 0;
}

/**
 * @brief Determine if a set of statstistics are in ascending order.
 */
__device__ bool is_ascending(const statistics_chunk* s,
                             Type ptype,
                             ConvertedType ctype,
                             uint32_t num_pages)
{
  for (uint32_t i = 1; i < num_pages; i++) {
    if (compare_values(ptype, ctype, s[i - 1].min_value, s[i].min_value) > 0 ||
        compare_values(ptype, ctype, s[i - 1].max_value, s[i].max_value) > 0) {
      return false;
    }
  }
  return true;
}

/**
 * @brief Determine if a set of statstistics are in descending order.
 */
__device__ bool is_descending(const statistics_chunk* s,
                              Type ptype,
                              ConvertedType ctype,
                              uint32_t num_pages)
{
  for (uint32_t i = 1; i < num_pages; i++) {
    if (compare_values(ptype, ctype, s[i - 1].min_value, s[i].min_value) < 0 ||
        compare_values(ptype, ctype, s[i - 1].max_value, s[i].max_value) < 0) {
      return false;
    }
  }
  return true;
}

/**
 * @brief Determine the ordering of a set of statistics.
 */
__device__ int32_t calculate_boundary_order(const statistics_chunk* s,
                                            Type ptype,
                                            ConvertedType ctype,
                                            uint32_t num_pages)
{
  if (not is_comparable(ptype, ctype)) { return BoundaryOrder::UNORDERED; }
  if (is_ascending(s, ptype, ctype, num_pages)) {
    return BoundaryOrder::ASCENDING;
  } else if (is_descending(s, ptype, ctype, num_pages)) {
    return BoundaryOrder::DESCENDING;
  }
  return BoundaryOrder::UNORDERED;
}

// align ptr to an 8-byte boundary. address returned will be <= ptr.
constexpr __device__ void* align8(void* ptr)
{
  // it's ok to round down because we have an extra 7 bytes in the buffer
  auto algn = 3 & reinterpret_cast<std::uintptr_t>(ptr);
  return static_cast<char*>(ptr) - algn;
}

}  // namespace

// blockDim(1, 1, 1)
__global__ void __launch_bounds__(1)
  gpuEncodeColumnIndexes(device_span<EncColumnChunk> chunks,
                         device_span<statistics_chunk const> column_stats,
                         int32_t column_index_truncate_length)
{
  __align__(8) unsigned char s_scratch[MIN_STATS_SCRATCH_SIZE];
  uint8_t* col_idx_end;

  if (column_stats.empty()) { return; }

  EncColumnChunk* ck_g             = &chunks[blockIdx.x];
  uint32_t num_pages               = ck_g->num_pages;
  parquet_column_device_view col_g = *ck_g->col_desc;
  size_t first_data_page           = ck_g->use_dictionary ? 1 : 0;
  uint32_t pageidx                 = ck_g->first_page;

  header_encoder encoder(ck_g->column_index_blob);

  // make sure scratch is aligned properly. here column_index_size indicates
  // how much scratch space is available for this chunk, including space for
  // truncation scratch + padding for alignment.
  void* scratch =
    column_index_truncate_length < MIN_STATS_SCRATCH_SIZE
      ? s_scratch
      : align8(ck_g->column_index_blob + ck_g->column_index_size - column_index_truncate_length);

  // null_pages
  encoder.field_list_begin(1, num_pages - first_data_page, ST_FLD_TRUE);
  for (uint32_t page = first_data_page; page < num_pages; page++) {
    encoder.put_bool(column_stats[pageidx + page].non_nulls == 0);
  }
  encoder.field_list_end(1);
  // min_values
  encoder.field_list_begin(2, num_pages - first_data_page, ST_FLD_BINARY);
  for (uint32_t page = first_data_page; page < num_pages; page++) {
    auto const [min_ptr, min_size] = get_extremum(&column_stats[pageidx + page].min_value,
                                                  col_g.stats_dtype,
                                                  scratch,
                                                  true,
                                                  column_index_truncate_length);
    encoder.put_binary(min_ptr, min_size);
  }
  encoder.field_list_end(2);
  // max_values
  encoder.field_list_begin(3, num_pages - first_data_page, ST_FLD_BINARY);
  for (uint32_t page = first_data_page; page < num_pages; page++) {
    auto const [max_ptr, max_size] = get_extremum(&column_stats[pageidx + page].max_value,
                                                  col_g.stats_dtype,
                                                  scratch,
                                                  false,
                                                  column_index_truncate_length);
    encoder.put_binary(max_ptr, max_size);
  }
  encoder.field_list_end(3);
  // boundary_order
  encoder.field_int32(4,
                      calculate_boundary_order(&column_stats[first_data_page + pageidx],
                                               col_g.physical_type,
                                               col_g.converted_type,
                                               num_pages - first_data_page));
  // null_counts
  encoder.field_list_begin(5, num_pages - first_data_page, ST_FLD_I64);
  for (uint32_t page = first_data_page; page < num_pages; page++) {
    encoder.put_int64(column_stats[pageidx + page].null_count);
  }
  encoder.field_list_end(5);
  encoder.end(&col_idx_end, false);

  // now reset column_index_size to the actual size of the encoded column index blob
  ck_g->column_index_size = static_cast<uint32_t>(col_idx_end - ck_g->column_index_blob);
}

void InitPageFragments(device_2dspan<PageFragment> frag,
                       device_span<parquet_column_device_view const> col_desc,
                       device_span<partition_info const> partitions,
                       device_span<int const> part_frag_offset,
                       uint32_t fragment_size,
                       rmm::cuda_stream_view stream)
{
  auto const num_columns              = frag.size().first;
  auto const num_fragments_per_column = frag.size().second;
  auto const grid_y = std::min(static_cast<uint32_t>(num_fragments_per_column), MAX_GRID_Y_SIZE);
  dim3 const dim_grid(num_columns, grid_y);  // 1 threadblock per fragment
  gpuInitPageFragments<512><<<dim_grid, 512, 0, stream.value()>>>(
    frag, col_desc, partitions, part_frag_offset, fragment_size);
}

void InitFragmentStatistics(device_2dspan<statistics_group> groups,
                            device_2dspan<PageFragment const> fragments,
                            device_span<parquet_column_device_view const> col_desc,
                            rmm::cuda_stream_view stream)
{
  int const num_columns              = col_desc.size();
  int const num_fragments_per_column = fragments.size().second;
  auto const y_dim =
    util::div_rounding_up_safe(num_fragments_per_column, 128 / cudf::detail::warp_size);
  auto const grid_y = std::min(static_cast<uint32_t>(y_dim), MAX_GRID_Y_SIZE);
  dim3 const dim_grid(num_columns, grid_y);  // 1 warp per fragment
  gpuInitFragmentStats<<<dim_grid, 128, 0, stream.value()>>>(groups, fragments, col_desc);
}

void InitEncoderPages(device_2dspan<EncColumnChunk> chunks,
                      device_span<gpu::EncPage> pages,
                      device_span<size_type> page_sizes,
                      device_span<size_type> comp_page_sizes,
                      device_span<parquet_column_device_view const> col_desc,
                      int32_t num_columns,
                      size_t max_page_size_bytes,
                      size_type max_page_size_rows,
                      uint32_t page_align,
                      statistics_merge_group* page_grstats,
                      statistics_merge_group* chunk_grstats,
                      rmm::cuda_stream_view stream)
{
  auto num_rowgroups = chunks.size().first;
  dim3 dim_grid(num_columns, num_rowgroups);  // 1 threadblock per rowgroup
  gpuInitPages<<<dim_grid, 128, 0, stream.value()>>>(chunks,
                                                     pages,
                                                     page_sizes,
                                                     comp_page_sizes,
                                                     col_desc,
                                                     page_grstats,
                                                     chunk_grstats,
                                                     num_columns,
                                                     max_page_size_bytes,
                                                     max_page_size_rows,
                                                     page_align);
}

void EncodePages(device_span<gpu::EncPage> pages,
                 device_span<device_span<uint8_t const>> comp_in,
                 device_span<device_span<uint8_t>> comp_out,
                 device_span<compression_result> comp_results,
                 rmm::cuda_stream_view stream)
{
  auto num_pages = pages.size();
  // A page is part of one column. This is launching 1 block per page. 1 block will exclusively
  // deal with one datatype.
  gpuEncodePages<128>
    <<<num_pages, 128, 0, stream.value()>>>(pages, comp_in, comp_out, comp_results);
}

void DecideCompression(device_span<EncColumnChunk> chunks, rmm::cuda_stream_view stream)
{
  gpuDecideCompression<<<chunks.size(), 128, 0, stream.value()>>>(chunks);
}

void EncodePageHeaders(device_span<EncPage> pages,
                       device_span<compression_result const> comp_results,
                       device_span<statistics_chunk const> page_stats,
                       const statistics_chunk* chunk_stats,
                       rmm::cuda_stream_view stream)
{
  // TODO: single thread task. No need for 128 threads/block. Earlier it used to employ rest of the
  // threads to coop load structs
  gpuEncodePageHeaders<<<pages.size(), 128, 0, stream.value()>>>(
    pages, comp_results, page_stats, chunk_stats);
}

void GatherPages(device_span<EncColumnChunk> chunks,
                 device_span<gpu::EncPage const> pages,
                 rmm::cuda_stream_view stream)
{
  gpuGatherPages<<<chunks.size(), 1024, 0, stream.value()>>>(chunks, pages);
}

void EncodeColumnIndexes(device_span<EncColumnChunk> chunks,
                         device_span<statistics_chunk const> column_stats,
                         int32_t column_index_truncate_length,
                         rmm::cuda_stream_view stream)
{
  gpuEncodeColumnIndexes<<<chunks.size(), 1, 0, stream.value()>>>(
    chunks, column_stats, column_index_truncate_length);
}

}  // namespace gpu
}  // namespace parquet
}  // namespace io
}  // namespace cudf
