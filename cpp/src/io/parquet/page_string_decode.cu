#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2018-2023, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "page_decode.cuh"
#include "page_string_utils.cuh"

#include <cudf/detail/utilities/cuda.cuh>
#include <cudf/strings/detail/gather.cuh>

namespace cudf::io::parquet::detail {

namespace {

constexpr int preprocess_block_size = 512;
constexpr int decode_block_size     = 128;
constexpr int rolling_buf_size      = decode_block_size * 2;
constexpr int preproc_buf_size      = LEVEL_DECODE_BUF_SIZE;

/**
 * @brief Compute the start and end page value bounds for this page
 *
 * This uses definition and repetition level info to determine the number of valid and null
 * values for the page, taking into account skip_rows/num_rows (if set).
 *
 * @param s The local page info
 * @param min_row Row index to start reading at
 * @param num_rows Maximum number of rows to read
 * @param is_bounds_pg True if this page is clipped
 * @param has_repetition True if the schema is nested
 * @param decoders Definition and repetition level decoders
 * @return pair containing start and end value indexes
 * @tparam level_t Type used to store decoded repetition and definition levels
 * @tparam rle_buf_size Size of the buffer used when decoding repetition and definition levels
 */
template <typename level_t, int rle_buf_size>
__device__ thrust::pair<int, int> page_bounds(page_state_s* const s,
                                              size_t min_row,
                                              size_t num_rows,
                                              bool is_bounds_pg,
                                              bool has_repetition,
                                              rle_stream<level_t, rle_buf_size>* decoders)
{
  using block_reduce = hipcub::BlockReduce<int, preprocess_block_size>;
  using block_scan   = hipcub::BlockScan<int, preprocess_block_size>;
  __shared__ union {
    typename block_reduce::TempStorage reduce_storage;
    typename block_scan::TempStorage scan_storage;
  } temp_storage;

  auto const t = threadIdx.x;

  // decode batches of level stream data using rle_stream objects and use the results to
  // calculate start and end value positions in the encoded string data.
  int const max_depth = s->col.max_nesting_depth;
  int const max_def   = s->nesting_info[max_depth - 1].max_def_level;

  // can skip all this if we know there are no nulls
  if (max_def == 0 && !is_bounds_pg) {
    s->page.num_valids = s->num_input_values;
    s->page.num_nulls  = 0;
    return {0, s->num_input_values};
  }

  int start_value = 0;
  int end_value   = s->page.num_input_values;
  auto const pp   = &s->page;
  auto const col  = &s->col;

  // initialize the stream decoders (requires values computed in setupLocalPageInfo)
  auto const def_decode = reinterpret_cast<level_t*>(pp->lvl_decode_buf[level_type::DEFINITION]);
  auto const rep_decode = reinterpret_cast<level_t*>(pp->lvl_decode_buf[level_type::REPETITION]);
  decoders[level_type::DEFINITION].init(s->col.level_bits[level_type::DEFINITION],
                                        s->abs_lvl_start[level_type::DEFINITION],
                                        s->abs_lvl_end[level_type::DEFINITION],
                                        preproc_buf_size,
                                        def_decode,
                                        s->page.num_input_values);
  // only need repetition if this is a bounds page. otherwise all we need is def level info
  // to count the nulls.
  if (has_repetition && is_bounds_pg) {
    decoders[level_type::REPETITION].init(s->col.level_bits[level_type::REPETITION],
                                          s->abs_lvl_start[level_type::REPETITION],
                                          s->abs_lvl_end[level_type::REPETITION],
                                          preproc_buf_size,
                                          rep_decode,
                                          s->page.num_input_values);
  }

  int processed = 0;

  // if this is a bounds page, we need to do extra work to find the start and/or end value index
  if (is_bounds_pg) {
    __shared__ int skipped_values;
    __shared__ int skipped_leaf_values;
    __shared__ int last_input_value;
    __shared__ int end_val_idx;

    // need these for skip_rows case
    auto const page_start_row = col->start_row + pp->chunk_row;
    auto const max_row        = min_row + num_rows;
    auto const begin_row      = page_start_row >= min_row ? 0 : min_row - page_start_row;
    auto const max_page_rows  = pp->num_rows - begin_row;
    auto const page_rows      = page_start_row + begin_row + max_page_rows <= max_row
                                  ? max_page_rows
                                  : max_row - (page_start_row + begin_row);
    auto end_row              = begin_row + page_rows;
    int row_fudge             = -1;

    // short circuit for no nulls
    if (max_def == 0 && !has_repetition) {
      if (t == 0) {
        pp->num_nulls  = 0;
        pp->num_valids = end_row - begin_row;
      }
      return {begin_row, end_row};
    }

    int row_count           = 0;
    int leaf_count          = 0;
    bool skipped_values_set = false;
    bool end_value_set      = false;

    while (processed < s->page.num_input_values) {
      thread_index_type start_val = processed;

      if (has_repetition) {
        decoders[level_type::REPETITION].decode_next(t);
        __syncthreads();

        // special case where page does not begin at a row boundary
        if (processed == 0 && rep_decode[0] != 0) {
          if (t == 0) {
            skipped_values      = 0;
            skipped_leaf_values = 0;
          }
          skipped_values_set = true;
          end_row++;  // need to finish off the previous row
          row_fudge = 0;
        }
      }

      // the # of rep/def levels will always be the same size
      processed += decoders[level_type::DEFINITION].decode_next(t);
      __syncthreads();

      // do something with the level data
      while (start_val < processed) {
        auto const idx_t = start_val + t;
        auto const idx   = rolling_index<preproc_buf_size>(idx_t);

        // get absolute thread row index
        int is_new_row = idx_t < processed && (!has_repetition || rep_decode[idx] == 0);
        int thread_row_count, block_row_count;
        block_scan(temp_storage.scan_storage)
          .InclusiveSum(is_new_row, thread_row_count, block_row_count);
        __syncthreads();

        // get absolute thread leaf index
        int const is_new_leaf = idx_t < processed && (def_decode[idx] >= max_def);
        int thread_leaf_count, block_leaf_count;
        block_scan(temp_storage.scan_storage)
          .InclusiveSum(is_new_leaf, thread_leaf_count, block_leaf_count);
        __syncthreads();

        // if we have not set skipped values yet, see if we found the first in-bounds row
        if (!skipped_values_set && row_count + block_row_count > begin_row) {
          // if this thread is in row bounds
          int const row_index = thread_row_count + row_count - 1;
          int const in_row_bounds =
            idx_t < processed && (row_index >= begin_row) && (row_index < end_row);

          int local_count, global_count;
          block_scan(temp_storage.scan_storage)
            .InclusiveSum(in_row_bounds, local_count, global_count);
          __syncthreads();

          // we found it
          if (global_count > 0) {
            // this is the thread that represents the first row. need to test in_row_bounds for
            // the case where we only want one row and local_count == 1 for many threads.
            if (local_count == 1 && in_row_bounds) {
              skipped_values = idx_t;
              skipped_leaf_values =
                leaf_count + (is_new_leaf ? thread_leaf_count - 1 : thread_leaf_count);
            }
            skipped_values_set = true;
          }
        }

        // test if row_count will exceed end_row in this batch
        if (!end_value_set && row_count + block_row_count >= end_row) {
          // if this thread exceeds row bounds. row_fudge change depending on whether we've faked
          // the end row to account for starting a page in the middle of a row.
          int const row_index          = thread_row_count + row_count + row_fudge;
          int const exceeds_row_bounds = row_index >= end_row;

          int local_count, global_count;
          block_scan(temp_storage.scan_storage)
            .InclusiveSum(exceeds_row_bounds, local_count, global_count);
          __syncthreads();

          // we found it
          if (global_count > 0) {
            // this is the thread that represents the end row.
            if (local_count == 1) {
              last_input_value = idx_t;
              end_val_idx = leaf_count + (is_new_leaf ? thread_leaf_count - 1 : thread_leaf_count);
            }
            end_value_set = true;
            break;
          }
        }

        row_count += block_row_count;
        leaf_count += block_leaf_count;

        start_val += preprocess_block_size;
      }
      __syncthreads();
      if (end_value_set) { break; }
    }

    start_value = skipped_values_set ? skipped_leaf_values : 0;
    end_value   = end_value_set ? end_val_idx : leaf_count;

    if (t == 0) {
      int const v0                = skipped_values_set ? skipped_values : 0;
      int const vn                = end_value_set ? last_input_value : s->num_input_values;
      int const total_values      = vn - v0;
      int const total_leaf_values = end_value - start_value;
      int const num_nulls         = total_values - total_leaf_values;
      pp->num_nulls               = num_nulls;
      pp->num_valids              = total_leaf_values;
    }
  }
  // already filtered out unwanted pages, so need to count all non-null values in this page
  else {
    int num_nulls = 0;
    while (processed < s->page.num_input_values) {
      thread_index_type start_val = processed;
      processed += decoders[level_type::DEFINITION].decode_next(t);
      __syncthreads();

      while (start_val < processed) {
        auto const idx_t = start_val + t;
        if (idx_t < processed) {
          auto const idx = rolling_index<preproc_buf_size>(idx_t);
          if (def_decode[idx] < max_def) { num_nulls++; }
        }
        start_val += preprocess_block_size;
      }
      __syncthreads();
    }

    int const null_count = block_reduce(temp_storage.reduce_storage).Sum(num_nulls);

    if (t == 0) {
      pp->num_nulls  = null_count;
      pp->num_valids = pp->num_input_values - null_count;
    }
    __syncthreads();

    end_value -= pp->num_nulls;
  }

  return {start_value, end_value};
}

/**
 * @brief Compute string size information for dictionary encoded strings.
 *
 * @param data Pointer to the start of the page data stream
 * @param dict_base Pointer to the start of the dictionary
 * @param dict_bits The number of bits used to in the dictionary bit packing
 * @param dict_size Size of the dictionary in bytes
 * @param data_size Size of the page data in bytes
 * @param start_value Do not count values that occur before this index
 * @param end_value Do not count values that occur after this index
 */
__device__ size_t totalDictEntriesSize(uint8_t const* data,
                                       uint8_t const* dict_base,
                                       int dict_bits,
                                       int dict_size,
                                       int data_size,
                                       int start_value,
                                       int end_value)
{
  int const t              = threadIdx.x;
  uint8_t const* ptr       = data;
  uint8_t const* const end = data + data_size;
  int const bytecnt        = (dict_bits + 7) >> 3;
  size_t l_str_len         = 0;  // partial sums across threads
  int pos                  = 0;  // current value index in the data stream
  int t0                   = 0;  // thread 0 for this batch

  int dict_run = 0;
  int dict_val = 0;

  while (pos < end_value && ptr <= end) {
    if (dict_run <= 1) {
      dict_run = (ptr < end) ? get_vlq32(ptr, end) : 0;
      if (!(dict_run & 1)) {
        // Repeated value
        if (ptr + bytecnt <= end) {
          int32_t run_val = ptr[0];
          if (bytecnt > 1) {
            run_val |= ptr[1] << 8;
            if (bytecnt > 2) {
              run_val |= ptr[2] << 16;
              if (bytecnt > 3) { run_val |= ptr[3] << 24; }
            }
          }
          dict_val = run_val & ((1 << dict_bits) - 1);
        }
        ptr += bytecnt;
      }
    }

    int batch_len;
    if (dict_run & 1) {
      // Literal batch: must output a multiple of 8, except for the last batch
      int batch_len_div8;
      batch_len      = max(min(preprocess_block_size, (int)(dict_run >> 1) * 8), 1);
      batch_len_div8 = (batch_len + 7) >> 3;
      dict_run -= batch_len_div8 * 2;
      ptr += batch_len_div8 * dict_bits;
    } else {
      batch_len = dict_run >> 1;
      dict_run  = 0;
    }

    int const is_literal = dict_run & 1;

    // calculate my thread id for this batch.  way to round-robin the work.
    int mytid = t - t0;
    if (mytid < 0) mytid += preprocess_block_size;

    // compute dictionary index.
    if (is_literal) {
      int dict_idx = 0;
      if (mytid < batch_len) {
        dict_idx         = dict_val;
        int32_t ofs      = (mytid - ((batch_len + 7) & ~7)) * dict_bits;
        const uint8_t* p = ptr + (ofs >> 3);
        ofs &= 7;
        if (p < end) {
          uint32_t c = 8 - ofs;
          dict_idx   = (*p++) >> ofs;
          if (c < dict_bits && p < end) {
            dict_idx |= (*p++) << c;
            c += 8;
            if (c < dict_bits && p < end) {
              dict_idx |= (*p++) << c;
              c += 8;
              if (c < dict_bits && p < end) { dict_idx |= (*p++) << c; }
            }
          }
          dict_idx &= (1 << dict_bits) - 1;
        }

        if (pos + mytid < end_value) {
          uint32_t const dict_pos = (dict_bits > 0) ? dict_idx * sizeof(string_index_pair) : 0;
          if (pos + mytid >= start_value && dict_pos < (uint32_t)dict_size) {
            const auto* src = reinterpret_cast<const string_index_pair*>(dict_base + dict_pos);
            l_str_len += src->second;
          }
        }
      }

      t0 += batch_len;
    } else {
      int const start_off =
        (pos < start_value && pos + batch_len > start_value) ? start_value - pos : 0;
      batch_len = min(batch_len, end_value - pos);
      if (mytid == 0) {
        uint32_t const dict_pos = (dict_bits > 0) ? dict_val * sizeof(string_index_pair) : 0;
        if (pos + batch_len > start_value && dict_pos < (uint32_t)dict_size) {
          const auto* src = reinterpret_cast<const string_index_pair*>(dict_base + dict_pos);
          l_str_len += (batch_len - start_off) * src->second;
        }
      }

      t0 += 1;
    }

    t0 = t0 % preprocess_block_size;
    pos += batch_len;
  }
  __syncthreads();

  using block_reduce = hipcub::BlockReduce<size_t, preprocess_block_size>;
  __shared__ typename block_reduce::TempStorage reduce_storage;
  size_t sum_l = block_reduce(reduce_storage).Sum(l_str_len);

  return sum_l;
}

/**
 * @brief Compute string size information for plain encoded strings.
 *
 * @param data Pointer to the start of the page data stream
 * @param data_size Length of data
 * @param start_value Do not count values that occur before this index
 * @param end_value Do not count values that occur after this index
 */
__device__ size_t totalPlainEntriesSize(uint8_t const* data,
                                        int data_size,
                                        int start_value,
                                        int end_value)
{
  int const t      = threadIdx.x;
  int pos          = 0;
  size_t total_len = 0;

  // This step is purely serial
  if (!t) {
    const uint8_t* cur = data;
    int k              = 0;

    while (pos < end_value && k < data_size) {
      int len;
      if (k + 4 <= data_size) {
        len = (cur[k]) | (cur[k + 1] << 8) | (cur[k + 2] << 16) | (cur[k + 3] << 24);
        k += 4;
        if (k + len > data_size) { len = 0; }
      } else {
        len = 0;
      }

      k += len;
      if (pos >= start_value) { total_len += len; }
      pos++;
    }
  }

  return total_len;
}

/**
 * @brief Kernel for computing string page output size information.
 *
 * String columns need accurate data size information to preallocate memory in the column buffer to
 * store the char data. This calls a kernel to calculate information needed by the string decoding
 * kernel. On exit, the `str_bytes`, `num_nulls`, and `num_valids` fields of the PageInfo struct
 * are updated. This call ignores non-string columns.
 *
 * @param pages All pages to be decoded
 * @param chunks All chunks to be decoded
 * @param min_rows crop all rows below min_row
 * @param num_rows Maximum number of rows to read
 * @tparam level_t Type used to store decoded repetition and definition levels
 */
template <typename level_t>
__global__ void __launch_bounds__(preprocess_block_size) gpuComputePageStringSizes(
  PageInfo* pages, device_span<ColumnChunkDesc const> chunks, size_t min_row, size_t num_rows)
{
  __shared__ __align__(16) page_state_s state_g;

  page_state_s* const s = &state_g;
  int const page_idx    = blockIdx.x;
  int const t           = threadIdx.x;
  PageInfo* const pp    = &pages[page_idx];

  // reset str_bytes to 0 in case it's already been calculated
  if (t == 0) { pp->str_bytes = 0; }

  // whether or not we have repetition levels (lists)
  bool const has_repetition = chunks[pp->chunk_idx].max_level[level_type::REPETITION] > 0;

  // the required number of runs in shared memory we will need to provide the
  // rle_stream object
  constexpr int rle_run_buffer_size = rle_stream_required_run_buffer_size<preprocess_block_size>();

  // the level stream decoders
  __shared__ rle_run<level_t> def_runs[rle_run_buffer_size];
  __shared__ rle_run<level_t> rep_runs[rle_run_buffer_size];
  rle_stream<level_t, preprocess_block_size> decoders[level_type::NUM_LEVEL_TYPES] = {{def_runs},
                                                                                      {rep_runs}};

  // setup page info
  if (!setupLocalPageInfo(
        s, pp, chunks, min_row, num_rows, mask_filter{KERNEL_MASK_STRING}, false)) {
    return;
  }

  if (!t) {
    s->page.num_nulls  = 0;
    s->page.num_valids = 0;
    s->page.str_bytes  = 0;
  }
  __syncthreads();

  bool const is_bounds_pg = is_bounds_page(s, min_row, num_rows, has_repetition);

  // if we're skipping this page anyway, no need to count it
  if (!is_bounds_pg && !is_page_contained(s, min_row, num_rows)) { return; }

  // find start/end value indices
  auto const [start_value, end_value] =
    page_bounds(s, min_row, num_rows, is_bounds_pg, has_repetition, decoders);

  // need to save num_nulls and num_valids calculated in page_bounds in this page
  if (t == 0) {
    pp->num_nulls  = s->page.num_nulls;
    pp->num_valids = s->page.num_valids;
  }

  auto const& col  = s->col;
  size_t str_bytes = 0;
  // short circuit for FIXED_LEN_BYTE_ARRAY
  if ((col.data_type & 7) == FIXED_LEN_BYTE_ARRAY) {
    str_bytes = pp->num_valids * s->dtype_len_in;
  } else {
    // now process string info in the range [start_value, end_value)
    // set up for decoding strings...can be either plain or dictionary
    uint8_t const* data      = s->data_start;
    uint8_t const* const end = s->data_end;
    uint8_t const* dict_base = nullptr;
    int dict_size            = 0;

    switch (pp->encoding) {
      case Encoding::PLAIN_DICTIONARY:
      case Encoding::RLE_DICTIONARY:
        // RLE-packed dictionary indices, first byte indicates index length in bits
        if (col.str_dict_index) {
          // String dictionary: use index
          dict_base = reinterpret_cast<const uint8_t*>(col.str_dict_index);
          dict_size = col.page_info[0].num_input_values * sizeof(string_index_pair);
        } else {
          dict_base = col.page_info[0].page_data;  // dictionary is always stored in the first page
          dict_size = col.page_info[0].uncompressed_page_size;
        }

        // FIXME: need to return an error condition...this won't actually do anything
        if (s->dict_bits > 32 || !dict_base) { CUDF_UNREACHABLE("invalid dictionary bit size"); }

        str_bytes = totalDictEntriesSize(
          data, dict_base, s->dict_bits, dict_size, (end - data), start_value, end_value);
        break;
      case Encoding::PLAIN:
        dict_size = static_cast<int32_t>(end - data);
        str_bytes = is_bounds_pg ? totalPlainEntriesSize(data, dict_size, start_value, end_value)
                                 : dict_size - sizeof(int) * pp->num_valids;
        break;
    }
  }

  if (t == 0) {
    // TODO check for overflow
    pp->str_bytes = str_bytes;
  }
}

/**
 * @brief Kernel for computing the string column data stored in the pages
 *
 * This function will write the page data and the page data's validity to the
 * output specified in the page's column chunk.
 *
 * This version uses a single warp to do the string copies.
 *
 * @param pages List of pages
 * @param chunks List of column chunks
 * @param min_row Row index to start reading at
 * @param num_rows Maximum number of rows to read
 * @tparam level_t Type used to store decoded repetition and definition levels
 */
template <typename level_t>
__global__ void __launch_bounds__(decode_block_size)
  gpuDecodeStringPageData(PageInfo* pages,
                          device_span<ColumnChunkDesc const> chunks,
                          size_t min_row,
                          size_t num_rows,
                          int32_t* error_code)
{
  __shared__ __align__(16) page_state_s state_g;
  __shared__ __align__(4) size_type last_offset;
  __shared__ __align__(16)
    page_state_buffers_s<rolling_buf_size, rolling_buf_size, rolling_buf_size>
      state_buffers;

  page_state_s* const s = &state_g;
  auto* const sb        = &state_buffers;
  int const page_idx    = blockIdx.x;
  int const t           = threadIdx.x;
  [[maybe_unused]] null_count_back_copier _{s, t};

  if (!setupLocalPageInfo(
        s, &pages[page_idx], chunks, min_row, num_rows, mask_filter{KERNEL_MASK_STRING}, true)) {
    return;
  }

  bool const has_repetition = s->col.max_level[level_type::REPETITION] > 0;

  // offsets are local to the page
  if (t == 0) { last_offset = 0; }
  __syncthreads();

  int const out_thread0                          = s->dict_base && s->dict_bits == 0 ? 32 : 64;
  int const leaf_level_index                     = s->col.max_nesting_depth - 1;
  PageNestingDecodeInfo* const nesting_info_base = s->nesting_info;

  __shared__ level_t rep[rolling_buf_size];  // circular buffer of repetition level values
  __shared__ level_t def[rolling_buf_size];  // circular buffer of definition level values

  // skipped_leaf_values will always be 0 for flat hierarchies.
  uint32_t skipped_leaf_values = s->page.skipped_leaf_values;
  while (s->error == 0 &&
         (s->input_value_count < s->num_input_values || s->src_pos < s->nz_count)) {
    int target_pos;
    int src_pos = s->src_pos;

    if (t < out_thread0) {
      target_pos = min(src_pos + 2 * (decode_block_size - out_thread0),
                       s->nz_count + (decode_block_size - out_thread0));
    } else {
      target_pos = min(s->nz_count, src_pos + decode_block_size - out_thread0);
      if (out_thread0 > 32) { target_pos = min(target_pos, s->dict_pos); }
    }
    __syncthreads();
    if (t < 32) {
      // decode repetition and definition levels.
      // - update validity vectors
      // - updates offsets (for nested columns)
      // - produces non-NULL value indices in s->nz_idx for subsequent decoding
      gpuDecodeLevels<rolling_buf_size, level_t>(s, sb, target_pos, rep, def, t);
    } else if (t < out_thread0) {
      // skipped_leaf_values will always be 0 for flat hierarchies.
      uint32_t src_target_pos = target_pos + skipped_leaf_values;

      // WARP1: Decode dictionary indices, booleans or string positions
      if (s->dict_base) {
        src_target_pos = gpuDecodeDictionaryIndices<false>(s, sb, src_target_pos, t & 0x1f).first;
      } else {
        gpuInitStringDescriptors<false>(s, sb, src_target_pos, t & 0x1f);
      }
      if (t == 32) { *(volatile int32_t*)&s->dict_pos = src_target_pos; }
    } else {
      int const me = t - out_thread0;

      // WARP1..WARP3: Decode values
      src_pos += t - out_thread0;

      // the position in the output column/buffer
      int dst_pos = sb->nz_idx[rolling_index<rolling_buf_size>(src_pos)];

      // for the flat hierarchy case we will be reading from the beginning of the value stream,
      // regardless of the value of first_row. so adjust our destination offset accordingly.
      // example:
      // - user has passed skip_rows = 2, so our first_row to output is 2
      // - the row values we get from nz_idx will be
      //   0, 1, 2, 3, 4 ....
      // - by shifting these values by first_row, the sequence becomes
      //   -1, -2, 0, 1, 2 ...
      // - so we will end up ignoring the first two input rows, and input rows 2..n will
      //   get written to the output starting at position 0.
      //
      if (!has_repetition) { dst_pos -= s->first_row; }

      // need to do this before we branch on src_pos/dst_pos so we don't deadlock
      // choose a character parallel string copy when the average string is longer than a warp
      using cudf::detail::warp_size;
      auto const use_char_ll =
        s->page.num_valids > 0 && (s->page.str_bytes / s->page.num_valids) >= warp_size;

      if (me < warp_size) {
        for (int i = 0; i < decode_block_size - out_thread0; i += warp_size) {
          dst_pos = sb->nz_idx[rolling_index<rolling_buf_size>(src_pos + i)];
          if (!has_repetition) { dst_pos -= s->first_row; }

          auto [ptr, len] = src_pos + i < target_pos && dst_pos >= 0
                              ? gpuGetStringData(s, sb, src_pos + skipped_leaf_values + i)
                              : cuda::std::pair<char const*, size_t>{nullptr, 0};

          __shared__ hipcub::WarpScan<size_type>::TempStorage temp_storage;
          size_type offset;
          hipcub::WarpScan<size_type>(temp_storage).ExclusiveSum(len, offset);
          offset += last_offset;

          if (use_char_ll) {
            __shared__ __align__(8) uint8_t const* pointers[warp_size];
            __shared__ __align__(4) size_type offsets[warp_size];
            __shared__ __align__(4) int dsts[warp_size];
            __shared__ __align__(4) int lengths[warp_size];

            offsets[me]  = offset;
            pointers[me] = reinterpret_cast<uint8_t const*>(ptr);
            dsts[me]     = dst_pos;
            lengths[me]  = len;
            __syncwarp();

            for (int ss = 0; ss < warp_size && ss + i + s->src_pos < target_pos; ss++) {
              if (dsts[ss] >= 0) {
                auto offptr =
                  reinterpret_cast<int32_t*>(nesting_info_base[leaf_level_index].data_out) +
                  dsts[ss];
                *offptr      = lengths[ss];
                auto str_ptr = nesting_info_base[leaf_level_index].string_out + offsets[ss];
                ll_strcpy(str_ptr, pointers[ss], lengths[ss], me);
              }
            }

          } else {
            if (src_pos + i < target_pos && dst_pos >= 0) {
              auto offptr =
                reinterpret_cast<int32_t*>(nesting_info_base[leaf_level_index].data_out) + dst_pos;
              *offptr      = len;
              auto str_ptr = nesting_info_base[leaf_level_index].string_out + offset;
              memcpy(str_ptr, ptr, len);
            }
            __syncwarp();
          }

          // last thread in warp updates last_offset
          if (me == warp_size - 1) { last_offset = offset + len; }
          __syncwarp();
        }
      }

      if (t == out_thread0) { *(volatile int32_t*)&s->src_pos = target_pos; }
    }
    __syncthreads();
  }

  // now turn array of lengths into offsets
  int value_count = nesting_info_base[leaf_level_index].value_count;

  // if no repetition we haven't calculated start/end bounds and instead just skipped
  // values until we reach first_row. account for that here.
  if (!has_repetition) { value_count -= s->first_row; }

  auto const offptr = reinterpret_cast<size_type*>(nesting_info_base[leaf_level_index].data_out);
  block_excl_sum<decode_block_size>(offptr, value_count, s->page.str_offset);

  if (t == 0 and s->error != 0) { set_error(s->error, error_code); }
}

}  // anonymous namespace

/**
 * @copydoc cudf::io::parquet::detail::ComputePageStringSizes
 */
void ComputePageStringSizes(cudf::detail::hostdevice_vector<PageInfo>& pages,
                            cudf::detail::hostdevice_vector<ColumnChunkDesc> const& chunks,
                            size_t min_row,
                            size_t num_rows,
                            int level_type_size,
                            rmm::cuda_stream_view stream)
{
  dim3 dim_block(preprocess_block_size, 1);
  dim3 dim_grid(pages.size(), 1);  // 1 threadblock per page
  if (level_type_size == 1) {
    gpuComputePageStringSizes<uint8_t>
      <<<dim_grid, dim_block, 0, stream.value()>>>(pages.device_ptr(), chunks, min_row, num_rows);
  } else {
    gpuComputePageStringSizes<uint16_t>
      <<<dim_grid, dim_block, 0, stream.value()>>>(pages.device_ptr(), chunks, min_row, num_rows);
  }
}

/**
 * @copydoc cudf::io::parquet::detail::DecodeStringPageData
 */
void __host__ DecodeStringPageData(cudf::detail::hostdevice_vector<PageInfo>& pages,
                                   cudf::detail::hostdevice_vector<ColumnChunkDesc> const& chunks,
                                   size_t num_rows,
                                   size_t min_row,
                                   int level_type_size,
                                   int32_t* error_code,
                                   rmm::cuda_stream_view stream)
{
  CUDF_EXPECTS(pages.size() > 0, "There is no page to decode");

  dim3 dim_block(decode_block_size, 1);
  dim3 dim_grid(pages.size(), 1);  // 1 threadblock per page

  if (level_type_size == 1) {
    gpuDecodeStringPageData<uint8_t><<<dim_grid, dim_block, 0, stream.value()>>>(
      pages.device_ptr(), chunks, min_row, num_rows, error_code);
  } else {
    gpuDecodeStringPageData<uint16_t><<<dim_grid, dim_block, 0, stream.value()>>>(
      pages.device_ptr(), chunks, min_row, num_rows, error_code);
  }
}

}  // namespace cudf::io::parquet::detail
