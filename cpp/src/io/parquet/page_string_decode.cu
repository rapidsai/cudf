#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2018-2024, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "delta_binary.cuh"
#include "error.hpp"
#include "page_decode.cuh"
#include "page_string_utils.cuh"
#include "rle_stream.cuh"

#include <cudf/detail/utilities/cuda.cuh>
#include <cudf/detail/utilities/stream_pool.hpp>
#include <cudf/strings/detail/gather.cuh>

#include <thrust/logical.h>
#include <thrust/transform_scan.h>

#include <bitset>

namespace cudf::io::parquet::detail {

namespace {

constexpr int preprocess_block_size    = 512;
constexpr int decode_block_size        = 128;
constexpr int delta_preproc_block_size = 64;
constexpr int delta_length_block_size  = 32;
constexpr int rolling_buf_size         = decode_block_size * 2;
constexpr int preproc_buf_size         = LEVEL_DECODE_BUF_SIZE;

/**
 * @brief Compute the start and end page value bounds for this page
 *
 * This uses definition and repetition level info to determine the number of valid and null
 * values for the page, taking into account skip_rows/num_rows (if set).
 *
 * @param s The local page info
 * @param min_row Row index to start reading at
 * @param num_rows Maximum number of rows to read
 * @param is_bounds_pg True if this page is clipped
 * @param has_repetition True if the schema is nested
 * @param decoders Definition and repetition level decoders
 * @return pair containing start and end value indexes
 * @tparam level_t Type used to store decoded repetition and definition levels
 * @tparam rle_buf_size Size of the buffer used when decoding repetition and definition levels
 */
template <typename level_t, int rle_buf_size>
__device__ thrust::pair<int, int> page_bounds(
  page_state_s* const s,
  size_t min_row,
  size_t num_rows,
  bool is_bounds_pg,
  bool has_repetition,
  rle_stream<level_t, rle_buf_size, preproc_buf_size>* decoders)
{
  using block_reduce = hipcub::BlockReduce<int, preprocess_block_size>;
  using block_scan   = hipcub::BlockScan<int, preprocess_block_size>;
  __shared__ union {
    typename block_reduce::TempStorage reduce_storage;
    typename block_scan::TempStorage scan_storage;
  } temp_storage;

  auto const t = threadIdx.x;

  // decode batches of level stream data using rle_stream objects and use the results to
  // calculate start and end value positions in the encoded string data.
  int const max_depth = s->col.max_nesting_depth;
  int const max_def   = s->nesting_info[max_depth - 1].max_def_level;

  // can skip all this if we know there are no nulls
  if (max_def == 0 && !is_bounds_pg) {
    if (t == 0) {
      s->page.num_valids = s->num_input_values;
      s->page.num_nulls  = 0;
    }
    return {0, s->num_input_values};
  }

  int start_value = 0;
  int end_value   = s->page.num_input_values;
  auto const pp   = &s->page;
  auto const col  = &s->col;

  // initialize the stream decoders (requires values computed in setupLocalPageInfo)
  auto const def_decode = reinterpret_cast<level_t*>(pp->lvl_decode_buf[level_type::DEFINITION]);
  auto const rep_decode = reinterpret_cast<level_t*>(pp->lvl_decode_buf[level_type::REPETITION]);
  decoders[level_type::DEFINITION].init(s->col.level_bits[level_type::DEFINITION],
                                        s->abs_lvl_start[level_type::DEFINITION],
                                        s->abs_lvl_end[level_type::DEFINITION],
                                        def_decode,
                                        s->page.num_input_values);
  // only need repetition if this is a bounds page. otherwise all we need is def level info
  // to count the nulls.
  if (has_repetition && is_bounds_pg) {
    decoders[level_type::REPETITION].init(s->col.level_bits[level_type::REPETITION],
                                          s->abs_lvl_start[level_type::REPETITION],
                                          s->abs_lvl_end[level_type::REPETITION],
                                          rep_decode,
                                          s->page.num_input_values);
  }

  int processed = 0;

  // if this is a bounds page, we need to do extra work to find the start and/or end value index
  if (is_bounds_pg) {
    __shared__ int skipped_values;
    __shared__ int skipped_leaf_values;
    __shared__ int last_input_value;
    __shared__ int end_val_idx;

    // need these for skip_rows case
    auto const page_start_row = col->start_row + pp->chunk_row;
    auto const max_row        = min_row + num_rows;
    auto const begin_row      = page_start_row >= min_row ? 0 : min_row - page_start_row;
    auto const max_page_rows  = pp->num_rows - begin_row;
    auto const page_rows      = page_start_row + begin_row + max_page_rows <= max_row
                                  ? max_page_rows
                                  : max_row - (page_start_row + begin_row);
    auto end_row              = begin_row + page_rows;
    int row_fudge             = -1;

    // short circuit for no nulls
    if (max_def == 0 && !has_repetition) {
      if (t == 0) {
        pp->num_nulls  = 0;
        pp->num_valids = end_row - begin_row;
      }
      return {begin_row, end_row};
    }

    int row_count           = 0;
    int leaf_count          = 0;
    bool skipped_values_set = false;
    bool end_value_set      = false;

    // If page_start_row >= min_row, then skipped_values is 0 and we don't have to search for
    // start_value. If there's repetition then we've already calculated
    // skipped_values/skipped_leaf_values.
    // TODO(ets): If we hit this condition, and end_row > last row in page, then we can skip
    // more of the processing below.
    if (has_repetition or page_start_row >= min_row) {
      if (t == 0) {
        if (has_repetition) {
          skipped_values      = pp->skipped_values;
          skipped_leaf_values = pp->skipped_leaf_values;
        } else {
          skipped_values      = 0;
          skipped_leaf_values = 0;
        }
      }
      skipped_values_set = true;
      __syncthreads();
    }

    while (processed < s->page.num_input_values) {
      thread_index_type start_val = processed;

      if (has_repetition) {
        decoders[level_type::REPETITION].decode_next(t);
        __syncthreads();

        // special case where page does not begin at a row boundary
        if (processed == 0 && rep_decode[0] != 0) {
          end_row++;  // need to finish off the previous row
          row_fudge = 0;
        }
      }

      // the # of rep/def levels will always be the same size
      processed += decoders[level_type::DEFINITION].decode_next(t);
      __syncthreads();

      // do something with the level data
      while (start_val < processed) {
        auto const idx_t = start_val + t;
        auto const idx   = rolling_index<preproc_buf_size>(idx_t);

        // get absolute thread row index
        int is_new_row = idx_t < processed && (!has_repetition || rep_decode[idx] == 0);
        int thread_row_count, block_row_count;
        block_scan(temp_storage.scan_storage)
          .InclusiveSum(is_new_row, thread_row_count, block_row_count);
        __syncthreads();

        // get absolute thread leaf index
        int const is_new_leaf = idx_t < processed && (def_decode[idx] >= max_def);
        int thread_leaf_count, block_leaf_count;
        block_scan(temp_storage.scan_storage)
          .InclusiveSum(is_new_leaf, thread_leaf_count, block_leaf_count);
        __syncthreads();

        // if we have not set skipped values yet, see if we found the first in-bounds row
        if (!skipped_values_set && row_count + block_row_count > begin_row) {
          // if this thread is in row bounds
          int const row_index = thread_row_count + row_count - 1;
          int const in_row_bounds =
            idx_t < processed && (row_index >= begin_row) && (row_index < end_row);

          int local_count, global_count;
          block_scan(temp_storage.scan_storage)
            .InclusiveSum(in_row_bounds, local_count, global_count);
          __syncthreads();

          // we found it
          if (global_count > 0) {
            // this is the thread that represents the first row. need to test in_row_bounds for
            // the case where we only want one row and local_count == 1 for many threads.
            if (local_count == 1 && in_row_bounds) {
              skipped_values = idx_t;
              skipped_leaf_values =
                leaf_count + (is_new_leaf ? thread_leaf_count - 1 : thread_leaf_count);
            }
            skipped_values_set = true;
          }
        }

        // test if row_count will exceed end_row in this batch
        if (!end_value_set && row_count + block_row_count >= end_row) {
          // if this thread exceeds row bounds. row_fudge change depending on whether we've faked
          // the end row to account for starting a page in the middle of a row.
          int const row_index          = thread_row_count + row_count + row_fudge;
          int const exceeds_row_bounds = row_index >= end_row;

          int local_count, global_count;
          block_scan(temp_storage.scan_storage)
            .InclusiveSum(exceeds_row_bounds, local_count, global_count);
          __syncthreads();

          // we found it
          if (global_count > 0) {
            // this is the thread that represents the end row.
            if (local_count == 1) {
              last_input_value = idx_t;
              end_val_idx = leaf_count + (is_new_leaf ? thread_leaf_count - 1 : thread_leaf_count);
            }
            end_value_set = true;
            break;
          }
        }

        row_count += block_row_count;
        leaf_count += block_leaf_count;

        start_val += preprocess_block_size;
      }
      __syncthreads();
      if (end_value_set) { break; }
    }

    start_value = skipped_values_set ? skipped_leaf_values : 0;
    end_value   = end_value_set ? end_val_idx : leaf_count;

    if (t == 0) {
      int const v0                = skipped_values_set ? skipped_values : 0;
      int const vn                = end_value_set ? last_input_value : s->num_input_values;
      int const total_values      = vn - v0;
      int const total_leaf_values = end_value - start_value;
      int const num_nulls         = total_values - total_leaf_values;
      pp->num_nulls               = num_nulls;
      pp->num_valids              = total_leaf_values;
    }
  }
  // already filtered out unwanted pages, so need to count all non-null values in this page
  else {
    int num_nulls = 0;
    while (processed < s->page.num_input_values) {
      thread_index_type start_val = processed;
      processed += decoders[level_type::DEFINITION].decode_next(t);
      __syncthreads();

      while (start_val < processed) {
        auto const idx_t = start_val + t;
        if (idx_t < processed) {
          auto const idx = rolling_index<preproc_buf_size>(idx_t);
          if (def_decode[idx] < max_def) { num_nulls++; }
        }
        start_val += preprocess_block_size;
      }
      __syncthreads();
    }

    int const null_count = block_reduce(temp_storage.reduce_storage).Sum(num_nulls);

    if (t == 0) {
      pp->num_nulls  = null_count;
      pp->num_valids = pp->num_input_values - null_count;
    }

    end_value -= pp->num_nulls;
  }

  return {start_value, end_value};
}

/**
 * @brief Compute string size information for dictionary encoded strings.
 *
 * @param data Pointer to the start of the page data stream
 * @param dict_base Pointer to the start of the dictionary
 * @param dict_bits The number of bits used to in the dictionary bit packing
 * @param dict_size Size of the dictionary in bytes
 * @param data_size Size of the page data in bytes
 * @param start_value Do not count values that occur before this index
 * @param end_value Do not count values that occur after this index
 */
__device__ size_t totalDictEntriesSize(uint8_t const* data,
                                       uint8_t const* dict_base,
                                       int dict_bits,
                                       int dict_size,
                                       int data_size,
                                       int start_value,
                                       int end_value)
{
  int const t              = threadIdx.x;
  uint8_t const* ptr       = data;
  uint8_t const* const end = data + data_size;
  int const bytecnt        = (dict_bits + 7) >> 3;
  size_t l_str_len         = 0;  // partial sums across threads
  int pos                  = 0;  // current value index in the data stream
  int t0                   = 0;  // thread 0 for this batch

  int dict_run = 0;
  int dict_val = 0;

  while (pos < end_value && ptr <= end) {
    if (dict_run <= 1) {
      dict_run = (ptr < end) ? get_vlq32(ptr, end) : 0;
      if (!(dict_run & 1)) {
        // Repeated value
        if (ptr + bytecnt <= end) {
          int32_t run_val = ptr[0];
          if (bytecnt > 1) {
            run_val |= ptr[1] << 8;
            if (bytecnt > 2) {
              run_val |= ptr[2] << 16;
              if (bytecnt > 3) { run_val |= ptr[3] << 24; }
            }
          }
          dict_val = run_val & ((1 << dict_bits) - 1);
        }
        ptr += bytecnt;
      }
    }

    int batch_len;
    if (dict_run & 1) {
      // Literal batch: must output a multiple of 8, except for the last batch
      int batch_len_div8;
      batch_len      = max(min(preprocess_block_size, (int)(dict_run >> 1) * 8), 1);
      batch_len_div8 = (batch_len + 7) >> 3;
      dict_run -= batch_len_div8 * 2;
      ptr += batch_len_div8 * dict_bits;
    } else {
      batch_len = dict_run >> 1;
      dict_run  = 0;
    }

    int const is_literal = dict_run & 1;

    // calculate my thread id for this batch.  way to round-robin the work.
    int mytid = t - t0;
    if (mytid < 0) mytid += preprocess_block_size;

    // compute dictionary index.
    if (is_literal) {
      int dict_idx = 0;
      if (mytid < batch_len) {
        dict_idx         = dict_val;
        int32_t ofs      = (mytid - ((batch_len + 7) & ~7)) * dict_bits;
        const uint8_t* p = ptr + (ofs >> 3);
        ofs &= 7;
        if (p < end) {
          uint32_t c = 8 - ofs;
          dict_idx   = (*p++) >> ofs;
          if (c < dict_bits && p < end) {
            dict_idx |= (*p++) << c;
            c += 8;
            if (c < dict_bits && p < end) {
              dict_idx |= (*p++) << c;
              c += 8;
              if (c < dict_bits && p < end) { dict_idx |= (*p++) << c; }
            }
          }
          dict_idx &= (1 << dict_bits) - 1;
        }

        if (pos + mytid < end_value) {
          uint32_t const dict_pos = (dict_bits > 0) ? dict_idx * sizeof(string_index_pair) : 0;
          if (pos + mytid >= start_value && dict_pos < (uint32_t)dict_size) {
            const auto* src = reinterpret_cast<const string_index_pair*>(dict_base + dict_pos);
            l_str_len += src->second;
          }
        }
      }

      t0 += batch_len;
    } else {
      int const start_off =
        (pos < start_value && pos + batch_len > start_value) ? start_value - pos : 0;
      batch_len = min(batch_len, end_value - pos);
      if (mytid == 0) {
        uint32_t const dict_pos = (dict_bits > 0) ? dict_val * sizeof(string_index_pair) : 0;
        if (pos + batch_len > start_value && dict_pos < (uint32_t)dict_size) {
          const auto* src = reinterpret_cast<const string_index_pair*>(dict_base + dict_pos);
          l_str_len += (batch_len - start_off) * src->second;
        }
      }

      t0 += 1;
    }

    t0 = t0 % preprocess_block_size;
    pos += batch_len;
  }
  __syncthreads();

  using block_reduce = hipcub::BlockReduce<size_t, preprocess_block_size>;
  __shared__ typename block_reduce::TempStorage reduce_storage;
  size_t sum_l = block_reduce(reduce_storage).Sum(l_str_len);

  return sum_l;
}

/**
 * @brief Compute string size information for plain encoded strings.
 *
 * @param data Pointer to the start of the page data stream
 * @param data_size Length of data
 * @param start_value Do not count values that occur before this index
 * @param end_value Do not count values that occur after this index
 */
__device__ size_t totalPlainEntriesSize(uint8_t const* data,
                                        int data_size,
                                        int start_value,
                                        int end_value)
{
  int const t      = threadIdx.x;
  int pos          = 0;
  size_t total_len = 0;

  // This step is purely serial
  if (!t) {
    const uint8_t* cur = data;
    int k              = 0;

    while (pos < end_value && k < data_size) {
      int len;
      if (k + 4 <= data_size) {
        len = (cur[k]) | (cur[k + 1] << 8) | (cur[k + 2] << 16) | (cur[k + 3] << 24);
        k += 4;
        if (k + len > data_size) { len = 0; }
      } else {
        len = 0;
      }

      k += len;
      if (pos >= start_value) { total_len += len; }
      pos++;
    }
  }

  return total_len;
}

/**
 * @brief Compute string size information for DELTA_BYTE_ARRAY encoded strings.
 *
 * This traverses the packed prefix and suffix lengths, summing them to obtain the total
 * number of bytes needed for the decoded string data. It also calculates an upper bound
 * for the largest string length to obtain an upper bound on temporary space needed if
 * rows will be skipped.
 *
 * Called with 64 threads.
 *
 * @param data Pointer to the start of the page data stream
 * @param end Pointer to the end of the page data stream
 * @param start_value Do not count values that occur before this index
 * @param end_value Do not count values that occur after this index
 * @return A pair of `size_t` values representing the total string size and temp buffer size
 * required for decoding
 */
__device__ thrust::pair<size_t, size_t> totalDeltaByteArraySize(uint8_t const* data,
                                                                uint8_t const* end,
                                                                int start_value,
                                                                int end_value)
{
  using cudf::detail::warp_size;
  using WarpReduce = hipcub::WarpReduce<uleb128_t>;
  __shared__ typename WarpReduce::TempStorage temp_storage[2];

  __shared__ __align__(16) delta_binary_decoder prefixes;
  __shared__ __align__(16) delta_binary_decoder suffixes;

  int const t       = threadIdx.x;
  int const lane_id = t % warp_size;
  int const warp_id = t / warp_size;

  if (t == 0) {
    auto const* suffix_start = prefixes.find_end_of_block(data, end);
    suffixes.init_binary_block(suffix_start, end);
  }
  __syncthreads();

  // two warps will traverse the prefixes and suffixes and sum them up
  auto const db = t < warp_size ? &prefixes : t < 2 * warp_size ? &suffixes : nullptr;

  size_t total_bytes = 0;
  uleb128_t max_len  = 0;

  if (db != nullptr) {
    // initialize with first value (which is stored in last_value)
    if (lane_id == 0 && start_value == 0) { total_bytes = db->last_value; }

    uleb128_t lane_sum = 0;
    uleb128_t lane_max = 0;
    while (db->current_value_idx < end_value &&
           db->current_value_idx < db->num_encoded_values(true)) {
      // calculate values for current mini-block
      db->calc_mini_block_values(lane_id);

      // get per lane sum for mini-block
      for (uint32_t i = 0; i < db->values_per_mb; i += 32) {
        uint32_t const idx = db->current_value_idx + i + lane_id;
        if (idx >= start_value && idx < end_value && idx < db->value_count) {
          lane_sum += db->value[rolling_index<delta_rolling_buf_size>(idx)];
        }
        // need lane_max over all values, not just in bounds
        if (idx < db->value_count) {
          lane_max = max(lane_max, db->value[rolling_index<delta_rolling_buf_size>(idx)]);
        }
      }

      if (lane_id == 0) { db->setup_next_mini_block(true); }
      __syncwarp();
    }

    // get sum for warp.
    // note: warp_sum will only be valid on lane 0.
    auto const warp_sum = WarpReduce(temp_storage[warp_id]).Sum(lane_sum);
    __syncwarp();
    auto const warp_max = WarpReduce(temp_storage[warp_id]).Reduce(lane_max, hipcub::Max());

    if (lane_id == 0) {
      total_bytes += warp_sum;
      max_len = warp_max;
    }
  }
  __syncthreads();

  // now sum up total_bytes from the two warps
  auto const final_bytes =
    cudf::detail::single_lane_block_sum_reduce<delta_preproc_block_size, 0>(total_bytes);

  // Sum up prefix and suffix max lengths to get a max possible string length. Multiply that
  // by the number of strings in a mini-block, plus one to save the last string.
  auto const temp_bytes =
    cudf::detail::single_lane_block_sum_reduce<delta_preproc_block_size, 0>(max_len) *
    (db->values_per_mb + 1);

  return {final_bytes, temp_bytes};
}

/**
 * @brief Kernel for computing string page bounds information.
 *
 * This kernel traverses the repetition and definition level data to determine start and end values
 * for pages with string-like data. Also calculates the number of null and valid values in the
 * page. Does nothing if the page mask is neither `STRING` nor `DELTA_BYTE_ARRAY`. On exit the
 * `num_nulls`, `num_valids`, `start_val` and `end_val` fields of the `PageInfo` struct will be
 * populated.
 *
 * @param pages All pages to be decoded
 * @param chunks All chunks to be decoded
 * @param min_rows crop all rows below min_row
 * @param num_rows Maximum number of rows to read
 * @tparam level_t Type used to store decoded repetition and definition levels
 */
template <typename level_t>
CUDF_KERNEL void __launch_bounds__(preprocess_block_size) gpuComputeStringPageBounds(
  PageInfo* pages, device_span<ColumnChunkDesc const> chunks, size_t min_row, size_t num_rows)
{
  __shared__ __align__(16) page_state_s state_g;

  page_state_s* const s = &state_g;
  int const page_idx    = blockIdx.x;
  int const t           = threadIdx.x;
  PageInfo* const pp    = &pages[page_idx];

  if (t == 0) {
    // don't clobber these if they're already computed from the index
    if (!pp->has_page_index) {
      s->page.num_nulls  = 0;
      s->page.num_valids = 0;
    }
    // reset str_bytes to 0 in case it's already been calculated (esp needed for chunked reads).
    pp->str_bytes = 0;
  }

  // whether or not we have repetition levels (lists)
  bool const has_repetition = chunks[pp->chunk_idx].max_level[level_type::REPETITION] > 0;

  // the required number of runs in shared memory we will need to provide the
  // rle_stream object
  constexpr int rle_run_buffer_size = rle_stream_required_run_buffer_size<preprocess_block_size>();

  // the level stream decoders
  __shared__ rle_run<level_t> def_runs[rle_run_buffer_size];
  __shared__ rle_run<level_t> rep_runs[rle_run_buffer_size];
  rle_stream<level_t, preprocess_block_size, preproc_buf_size>
    decoders[level_type::NUM_LEVEL_TYPES] = {{def_runs}, {rep_runs}};

  // setup page info
  if (!setupLocalPageInfo(s,
                          pp,
                          chunks,
                          min_row,
                          num_rows,
                          mask_filter{STRINGS_MASK},
                          page_processing_stage::STRING_BOUNDS)) {
    return;
  }

  bool const is_bounds_pg = is_bounds_page(s, min_row, num_rows, has_repetition);

  // if we have size info, then we only need to do this for bounds pages
  if (pp->has_page_index && !is_bounds_pg) { return; }

  // find start/end value indices
  auto const [start_value, end_value] =
    page_bounds(s, min_row, num_rows, is_bounds_pg, has_repetition, decoders);

  // need to save num_nulls and num_valids calculated in page_bounds in this page
  if (t == 0) {
    pp->num_nulls  = s->page.num_nulls;
    pp->num_valids = s->page.num_valids;
    pp->start_val  = start_value;
    pp->end_val    = end_value;
  }
}

/**
 * @brief Kernel for computing string page output size information for delta_byte_array encoding.
 *
 * This call ignores columns that are not DELTA_BYTE_ARRAY encoded. On exit the `str_bytes` field
 * of the `PageInfo` struct will be populated. Also fills in the `temp_string_size` field if rows
 * are to be skipped.
 *
 * @param pages All pages to be decoded
 * @param chunks All chunks to be decoded
 * @param min_rows crop all rows below min_row
 * @param num_rows Maximum number of rows to read
 */
CUDF_KERNEL void __launch_bounds__(delta_preproc_block_size) gpuComputeDeltaPageStringSizes(
  PageInfo* pages, device_span<ColumnChunkDesc const> chunks, size_t min_row, size_t num_rows)
{
  __shared__ __align__(16) page_state_s state_g;

  page_state_s* const s = &state_g;
  int const page_idx    = blockIdx.x;
  int const t           = threadIdx.x;
  PageInfo* const pp    = &pages[page_idx];

  // whether or not we have repetition levels (lists)
  bool const has_repetition = chunks[pp->chunk_idx].max_level[level_type::REPETITION] > 0;

  // setup page info
  if (!setupLocalPageInfo(s,
                          pp,
                          chunks,
                          min_row,
                          num_rows,
                          mask_filter{decode_kernel_mask::DELTA_BYTE_ARRAY},
                          page_processing_stage::STRING_BOUNDS)) {
    return;
  }

  auto const start_value = pp->start_val;

  // if data size is known, can short circuit here
  if ((chunks[pp->chunk_idx].data_type & 7) == FIXED_LEN_BYTE_ARRAY) {
    if (t == 0) {
      pp->str_bytes = pp->num_valids * s->dtype_len_in;

      // only need temp space if we're skipping values
      if (start_value > 0) {
        // just need to parse the header of the first delta binary block to get values_per_mb
        delta_binary_decoder db;
        db.init_binary_block(s->data_start, s->data_end);
        // save enough for one mini-block plus some extra to save the last_string
        pp->temp_string_size = s->dtype_len_in * (db.values_per_mb + 1);
      }
    }
  } else {
    bool const is_bounds_pg = is_bounds_page(s, min_row, num_rows, has_repetition);

    // if we have size info, then we only need to do this for bounds pages
    if (pp->has_page_index && !is_bounds_pg) {
      // check if we need to store values from the index
      if (is_page_contained(s, min_row, num_rows)) { pp->str_bytes = pp->str_bytes_from_index; }
      return;
    }

    // now process string info in the range [start_value, end_value)
    // set up for decoding strings...can be either plain or dictionary
    uint8_t const* data      = s->data_start;
    uint8_t const* const end = s->data_end;
    auto const end_value     = pp->end_val;

    auto const [len, temp_bytes] = totalDeltaByteArraySize(data, end, start_value, end_value);

    if (t == 0) {
      pp->str_bytes = len;

      // only need temp space if we're skipping values
      if (start_value > 0) { pp->temp_string_size = temp_bytes; }
    }
  }
}

/**
 * @brief Kernel for computing string page output size information for DELTA_LENGTH_BYTE_ARRAY
 * encoding.
 *
 * This call ignores columns that are not DELTA_LENGTH_BYTE_ARRAY encoded. On exit the `str_bytes`
 * field of the `PageInfo` struct will be populated.
 *
 * Currently this function only supports being called by a single warp.
 *
 * @param pages All pages to be decoded
 * @param chunks All chunks to be decoded
 * @param min_rows crop all rows below min_row
 * @param num_rows Maximum number of rows to read
 */
CUDF_KERNEL void __launch_bounds__(delta_length_block_size) gpuComputeDeltaLengthPageStringSizes(
  PageInfo* pages, device_span<ColumnChunkDesc const> chunks, size_t min_row, size_t num_rows)
{
  using cudf::detail::warp_size;
  using WarpReduce = hipcub::WarpReduce<uleb128_t>;
  __shared__ typename WarpReduce::TempStorage temp_storage;
  __shared__ __align__(16) page_state_s state_g;
  __shared__ __align__(16) delta_binary_decoder string_lengths;

  page_state_s* const s = &state_g;
  int const page_idx    = blockIdx.x;
  int const t           = threadIdx.x;
  PageInfo* const pp    = &pages[page_idx];

  // whether or not we have repetition levels (lists)
  bool const has_repetition = chunks[pp->chunk_idx].max_level[level_type::REPETITION] > 0;

  // setup page info
  if (!setupLocalPageInfo(s,
                          pp,
                          chunks,
                          min_row,
                          num_rows,
                          mask_filter{decode_kernel_mask::DELTA_LENGTH_BA},
                          page_processing_stage::STRING_BOUNDS)) {
    return;
  }

  bool const is_bounds_pg = is_bounds_page(s, min_row, num_rows, has_repetition);

  // if we have size info, then we only need to do this for bounds pages
  if (pp->has_page_index && !is_bounds_pg) {
    // check if we need to store values from the index
    if (is_page_contained(s, min_row, num_rows)) { pp->str_bytes = pp->str_bytes_from_index; }
    return;
  }

  // for DELTA_LENGTH_BYTE_ARRAY, string size is page_data_size - size_of_delta_binary_block.
  // so all we need to do is skip the encoded string size info and then do pointer arithmetic,
  // if this isn't a bounds page.
  if (not is_bounds_pg) {
    if (t == 0) {
      auto const* string_start = string_lengths.find_end_of_block(s->data_start, s->data_end);
      size_t len               = static_cast<size_t>(s->data_end - string_start);
      pp->str_bytes            = len;
    }
  } else {
    // now process string info in the range [start_value, end_value)
    // set up for decoding strings...can be either plain or dictionary
    auto const start_value = pp->start_val;
    auto const end_value   = pp->end_val;

    if (t == 0) { string_lengths.init_binary_block(s->data_start, s->data_end); }
    __syncwarp();

    size_t total_bytes = 0;

    // initialize with first value (unless there are no values)
    if (t == 0 && start_value == 0 && start_value < end_value) {
      total_bytes = string_lengths.value_at(0);
    }

    uleb128_t lane_sum = 0;
    while (string_lengths.current_value_idx < end_value &&
           string_lengths.current_value_idx < string_lengths.num_encoded_values(true)) {
      // calculate values for current mini-block
      string_lengths.calc_mini_block_values(t);

      // get per lane sum for mini-block
      for (uint32_t i = 0; i < string_lengths.values_per_mb; i += warp_size) {
        uint32_t const idx = string_lengths.current_value_idx + i + t;
        if (idx >= start_value && idx < end_value && idx < string_lengths.value_count) {
          lane_sum += string_lengths.value[rolling_index<delta_rolling_buf_size>(idx)];
        }
      }

      if (t == 0) { string_lengths.setup_next_mini_block(true); }
      __syncwarp();
    }

    // get sum for warp.
    // note: warp_sum will only be valid on lane 0.
    auto const warp_sum = WarpReduce(temp_storage).Sum(lane_sum);

    if (t == 0) {
      total_bytes += warp_sum;
      pp->str_bytes = total_bytes;
    }
  }
}

/**
 * @brief Kernel for computing string page output size information.
 *
 * This call ignores non-string columns. On exit the `str_bytes` field of the `PageInfo` struct will
 * be populated.
 *
 * @param pages All pages to be decoded
 * @param chunks All chunks to be decoded
 * @param min_rows crop all rows below min_row
 * @param num_rows Maximum number of rows to read
 */
CUDF_KERNEL void __launch_bounds__(preprocess_block_size) gpuComputePageStringSizes(
  PageInfo* pages, device_span<ColumnChunkDesc const> chunks, size_t min_row, size_t num_rows)
{
  __shared__ __align__(16) page_state_s state_g;

  page_state_s* const s = &state_g;
  int const page_idx    = blockIdx.x;
  int const t           = threadIdx.x;
  PageInfo* const pp    = &pages[page_idx];

  // whether or not we have repetition levels (lists)
  bool const has_repetition = chunks[pp->chunk_idx].max_level[level_type::REPETITION] > 0;

  // setup page info
  if (!setupLocalPageInfo(s,
                          pp,
                          chunks,
                          min_row,
                          num_rows,
                          mask_filter{decode_kernel_mask::STRING},
                          page_processing_stage::STRING_BOUNDS)) {
    return;
  }

  bool const is_bounds_pg = is_bounds_page(s, min_row, num_rows, has_repetition);

  // if we have size info, then we only need to do this for bounds pages
  if (pp->has_page_index && !is_bounds_pg) {
    // check if we need to store values from the index
    if (is_page_contained(s, min_row, num_rows)) { pp->str_bytes = pp->str_bytes_from_index; }
    return;
  }

  auto const& col  = s->col;
  size_t str_bytes = 0;
  // short circuit for FIXED_LEN_BYTE_ARRAY
  if ((col.data_type & 7) == FIXED_LEN_BYTE_ARRAY) {
    str_bytes = pp->num_valids * s->dtype_len_in;
  } else {
    // now process string info in the range [start_value, end_value)
    // set up for decoding strings...can be either plain or dictionary
    uint8_t const* data      = s->data_start;
    uint8_t const* const end = s->data_end;
    uint8_t const* dict_base = nullptr;
    int dict_size            = 0;
    auto const start_value   = pp->start_val;
    auto const end_value     = pp->end_val;

    switch (pp->encoding) {
      case Encoding::PLAIN_DICTIONARY:
      case Encoding::RLE_DICTIONARY:
        // RLE-packed dictionary indices, first byte indicates index length in bits
        if (col.str_dict_index) {
          // String dictionary: use index
          dict_base = reinterpret_cast<const uint8_t*>(col.str_dict_index);
          dict_size = col.dict_page->num_input_values * sizeof(string_index_pair);
        } else {
          dict_base = col.dict_page->page_data;
          dict_size = col.dict_page->uncompressed_page_size;
        }

        // FIXME: need to return an error condition...this won't actually do anything
        if (s->dict_bits > 32 || (!dict_base && col.dict_page->num_input_values > 0)) {
          CUDF_UNREACHABLE("invalid dictionary bit size");
        }

        str_bytes = totalDictEntriesSize(
          data, dict_base, s->dict_bits, dict_size, (end - data), start_value, end_value);
        break;
      case Encoding::PLAIN:
        dict_size = static_cast<int32_t>(end - data);
        str_bytes = is_bounds_pg ? totalPlainEntriesSize(data, dict_size, start_value, end_value)
                                 : dict_size - sizeof(int) * pp->num_valids;
        break;
    }
  }

  if (t == 0) {
    // TODO check for overflow
    pp->str_bytes = str_bytes;

    // only need temp space for delta
    pp->temp_string_size = 0;
  }
}

/**
 * @brief Kernel for computing the string column data stored in the pages
 *
 * This function will write the page data and the page data's validity to the
 * output specified in the page's column chunk.
 *
 * This version uses a single warp to do the string copies.
 *
 * @param pages List of pages
 * @param chunks List of column chunks
 * @param min_row Row index to start reading at
 * @param num_rows Maximum number of rows to read
 * @tparam level_t Type used to store decoded repetition and definition levels
 */
template <typename level_t>
CUDF_KERNEL void __launch_bounds__(decode_block_size)
  gpuDecodeStringPageData(PageInfo* pages,
                          device_span<ColumnChunkDesc const> chunks,
                          size_t min_row,
                          size_t num_rows,
                          kernel_error::pointer error_code)
{
  using cudf::detail::warp_size;
  __shared__ __align__(16) page_state_s state_g;
  __shared__ __align__(4) size_type last_offset;
  __shared__ __align__(16)
    page_state_buffers_s<rolling_buf_size, rolling_buf_size, rolling_buf_size>
      state_buffers;

  page_state_s* const s = &state_g;
  auto* const sb        = &state_buffers;
  int const page_idx    = blockIdx.x;
  int const t           = threadIdx.x;
  int const lane_id     = t % warp_size;
  [[maybe_unused]] null_count_back_copier _{s, t};

  if (!setupLocalPageInfo(s,
                          &pages[page_idx],
                          chunks,
                          min_row,
                          num_rows,
                          mask_filter{decode_kernel_mask::STRING},
                          page_processing_stage::DECODE)) {
    return;
  }

  bool const has_repetition = s->col.max_level[level_type::REPETITION] > 0;

  // offsets are local to the page
  if (t == 0) { last_offset = 0; }
  __syncthreads();

  int const out_thread0                          = s->dict_base && s->dict_bits == 0 ? 32 : 64;
  int const leaf_level_index                     = s->col.max_nesting_depth - 1;
  PageNestingDecodeInfo* const nesting_info_base = s->nesting_info;

  __shared__ level_t rep[rolling_buf_size];  // circular buffer of repetition level values
  __shared__ level_t def[rolling_buf_size];  // circular buffer of definition level values

  // skipped_leaf_values will always be 0 for flat hierarchies.
  uint32_t skipped_leaf_values = s->page.skipped_leaf_values;
  while (s->error == 0 &&
         (s->input_value_count < s->num_input_values || s->src_pos < s->nz_count)) {
    int target_pos;
    int src_pos = s->src_pos;

    if (t < out_thread0) {
      target_pos = min(src_pos + 2 * (decode_block_size - out_thread0),
                       s->nz_count + (decode_block_size - out_thread0));
    } else {
      target_pos = min(s->nz_count, src_pos + decode_block_size - out_thread0);
      if (out_thread0 > 32) { target_pos = min(target_pos, s->dict_pos); }
    }
    // this needs to be here to prevent warp 1/2 modifying src_pos before all threads have read it
    __syncthreads();
    if (t < 32) {
      // decode repetition and definition levels.
      // - update validity vectors
      // - updates offsets (for nested columns)
      // - produces non-NULL value indices in s->nz_idx for subsequent decoding
      gpuDecodeLevels<rolling_buf_size, level_t>(s, sb, target_pos, rep, def, t);
    } else if (t < out_thread0) {
      // skipped_leaf_values will always be 0 for flat hierarchies.
      uint32_t src_target_pos = target_pos + skipped_leaf_values;

      // WARP1: Decode dictionary indices, booleans or string positions
      if (s->dict_base) {
        src_target_pos = gpuDecodeDictionaryIndices<false>(s, sb, src_target_pos, lane_id).first;
      } else {
        gpuInitStringDescriptors<false>(s, sb, src_target_pos, lane_id);
      }
      if (t == 32) { s->dict_pos = src_target_pos; }
    } else {
      int const me = t - out_thread0;

      // WARP1..WARP3: Decode values
      src_pos += t - out_thread0;

      // the position in the output column/buffer
      int dst_pos = sb->nz_idx[rolling_index<rolling_buf_size>(src_pos)];

      // for the flat hierarchy case we will be reading from the beginning of the value stream,
      // regardless of the value of first_row. so adjust our destination offset accordingly.
      // example:
      // - user has passed skip_rows = 2, so our first_row to output is 2
      // - the row values we get from nz_idx will be
      //   0, 1, 2, 3, 4 ....
      // - by shifting these values by first_row, the sequence becomes
      //   -1, -2, 0, 1, 2 ...
      // - so we will end up ignoring the first two input rows, and input rows 2..n will
      //   get written to the output starting at position 0.
      //
      if (!has_repetition) { dst_pos -= s->first_row; }

      if (me < warp_size) {
        for (int i = 0; i < decode_block_size - out_thread0; i += warp_size) {
          dst_pos = sb->nz_idx[rolling_index<rolling_buf_size>(src_pos + i)];
          if (!has_repetition) { dst_pos -= s->first_row; }

          auto [ptr, len] = src_pos + i < target_pos && dst_pos >= 0
                              ? gpuGetStringData(s, sb, src_pos + skipped_leaf_values + i)
                              : cuda::std::pair<char const*, size_t>{nullptr, 0};

          __shared__ hipcub::WarpScan<size_type>::TempStorage temp_storage;
          size_type offset, warp_total;
          hipcub::WarpScan<size_type>(temp_storage).ExclusiveSum(len, offset, warp_total);
          offset += last_offset;

          // choose a character parallel string copy when the average string is longer than a warp
          auto const use_char_ll = warp_total / warp_size >= warp_size;

          if (use_char_ll) {
            __shared__ __align__(8) uint8_t const* pointers[warp_size];
            __shared__ __align__(4) size_type offsets[warp_size];
            __shared__ __align__(4) int dsts[warp_size];
            __shared__ __align__(4) int lengths[warp_size];

            offsets[me]  = offset;
            pointers[me] = reinterpret_cast<uint8_t const*>(ptr);
            dsts[me]     = dst_pos;
            lengths[me]  = len;
            __syncwarp();

            for (int ss = 0; ss < warp_size && ss + i + s->src_pos < target_pos; ss++) {
              if (dsts[ss] >= 0) {
                auto offptr =
                  reinterpret_cast<int32_t*>(nesting_info_base[leaf_level_index].data_out) +
                  dsts[ss];
                *offptr      = lengths[ss];
                auto str_ptr = nesting_info_base[leaf_level_index].string_out + offsets[ss];
                ll_strcpy(str_ptr, pointers[ss], lengths[ss], me);
              }
            }

          } else {
            if (src_pos + i < target_pos && dst_pos >= 0) {
              auto offptr =
                reinterpret_cast<int32_t*>(nesting_info_base[leaf_level_index].data_out) + dst_pos;
              *offptr      = len;
              auto str_ptr = nesting_info_base[leaf_level_index].string_out + offset;
              memcpy(str_ptr, ptr, len);
            }
            __syncwarp();
          }

          // last thread in warp updates last_offset
          if (me == warp_size - 1) { last_offset = offset + len; }
          __syncwarp();
        }
      }

      if (t == out_thread0) { s->src_pos = target_pos; }
    }
    __syncthreads();
  }

  // now turn array of lengths into offsets
  int value_count = nesting_info_base[leaf_level_index].value_count;

  // if no repetition we haven't calculated start/end bounds and instead just skipped
  // values until we reach first_row. account for that here.
  if (!has_repetition) { value_count -= s->first_row; }

  auto const offptr = reinterpret_cast<size_type*>(nesting_info_base[leaf_level_index].data_out);
  block_excl_sum<decode_block_size>(offptr, value_count, s->page.str_offset);

  if (t == 0 and s->error != 0) { set_error(s->error, error_code); }
}

// Functor used to set the `temp_string_buf` pointer for each page. `data` points to a buffer
// to be used when skipping rows in the delta_byte_array decoder. Given a page and an offset,
// set the page's `temp_string_buf` to be `data + offset`.
struct page_tform_functor {
  uint8_t* const data;

  __device__ PageInfo operator()(PageInfo& page, int64_t offset)
  {
    if (page.temp_string_size != 0) { page.temp_string_buf = data + offset; }
    return page;
  }
};

}  // anonymous namespace

/**
 * @copydoc cudf::io::parquet::detail::ComputePageStringSizes
 */
void ComputePageStringSizes(cudf::detail::hostdevice_span<PageInfo> pages,
                            cudf::detail::hostdevice_span<ColumnChunkDesc const> chunks,
                            rmm::device_uvector<uint8_t>& temp_string_buf,
                            size_t min_row,
                            size_t num_rows,
                            int level_type_size,
                            uint32_t kernel_mask,
                            rmm::cuda_stream_view stream)
{
  dim3 const dim_block(preprocess_block_size, 1);
  dim3 const dim_grid(pages.size(), 1);  // 1 threadblock per page
  if (level_type_size == 1) {
    gpuComputeStringPageBounds<uint8_t>
      <<<dim_grid, dim_block, 0, stream.value()>>>(pages.device_ptr(), chunks, min_row, num_rows);
  } else {
    gpuComputeStringPageBounds<uint16_t>
      <<<dim_grid, dim_block, 0, stream.value()>>>(pages.device_ptr(), chunks, min_row, num_rows);
  }

  // kernel mask may contain other kernels we don't need to count
  int const count_mask = kernel_mask & STRINGS_MASK;
  int const nkernels   = std::bitset<32>(count_mask).count();
  auto const streams   = cudf::detail::fork_streams(stream, nkernels);

  int s_idx = 0;
  if (BitAnd(kernel_mask, decode_kernel_mask::DELTA_BYTE_ARRAY) != 0) {
    dim3 dim_delta(delta_preproc_block_size, 1);
    gpuComputeDeltaPageStringSizes<<<dim_grid, dim_delta, 0, streams[s_idx++].value()>>>(
      pages.device_ptr(), chunks, min_row, num_rows);
  }
  if (BitAnd(kernel_mask, decode_kernel_mask::DELTA_LENGTH_BA) != 0) {
    dim3 dim_delta(delta_length_block_size, 1);
    gpuComputeDeltaLengthPageStringSizes<<<dim_grid, dim_delta, 0, streams[s_idx++].value()>>>(
      pages.device_ptr(), chunks, min_row, num_rows);
  }
  if (BitAnd(kernel_mask, decode_kernel_mask::STRING) != 0) {
    gpuComputePageStringSizes<<<dim_grid, dim_block, 0, streams[s_idx++].value()>>>(
      pages.device_ptr(), chunks, min_row, num_rows);
  }

  // synchronize the streams
  cudf::detail::join_streams(streams, stream);

  // check for needed temp space for DELTA_BYTE_ARRAY
  auto const need_sizes = thrust::any_of(
    rmm::exec_policy(stream), pages.device_begin(), pages.device_end(), [] __device__(auto& page) {
      return page.temp_string_size != 0;
    });

  if (need_sizes) {
    // sum up all of the temp_string_sizes
    auto const page_sizes = [] __device__(PageInfo const& page) { return page.temp_string_size; };
    auto const total_size = thrust::transform_reduce(rmm::exec_policy(stream),
                                                     pages.device_begin(),
                                                     pages.device_end(),
                                                     page_sizes,
                                                     0L,
                                                     thrust::plus<int64_t>{});

    // now do an exclusive scan over the temp_string_sizes to get offsets for each
    // page's chunk of the temp buffer
    rmm::device_uvector<int64_t> page_string_offsets(pages.size(), stream);
    thrust::transform_exclusive_scan(rmm::exec_policy_nosync(stream),
                                     pages.device_begin(),
                                     pages.device_end(),
                                     page_string_offsets.begin(),
                                     page_sizes,
                                     0L,
                                     thrust::plus<int64_t>{});

    // allocate the temp space
    temp_string_buf.resize(total_size, stream);

    // now use the offsets array to set each page's temp_string_buf pointers
    thrust::transform(rmm::exec_policy_nosync(stream),
                      pages.device_begin(),
                      pages.device_end(),
                      page_string_offsets.begin(),
                      pages.device_begin(),
                      page_tform_functor{temp_string_buf.data()});
  }
}

/**
 * @copydoc cudf::io::parquet::detail::DecodeStringPageData
 */
void __host__ DecodeStringPageData(cudf::detail::hostdevice_span<PageInfo> pages,
                                   cudf::detail::hostdevice_span<ColumnChunkDesc const> chunks,
                                   size_t num_rows,
                                   size_t min_row,
                                   int level_type_size,
                                   kernel_error::pointer error_code,
                                   rmm::cuda_stream_view stream)
{
  CUDF_EXPECTS(pages.size() > 0, "There is no page to decode");

  dim3 dim_block(decode_block_size, 1);
  dim3 dim_grid(pages.size(), 1);  // 1 threadblock per page

  if (level_type_size == 1) {
    gpuDecodeStringPageData<uint8_t><<<dim_grid, dim_block, 0, stream.value()>>>(
      pages.device_ptr(), chunks, min_row, num_rows, error_code);
  } else {
    gpuDecodeStringPageData<uint16_t><<<dim_grid, dim_block, 0, stream.value()>>>(
      pages.device_ptr(), chunks, min_row, num_rows, error_code);
  }
}

}  // namespace cudf::io::parquet::detail
