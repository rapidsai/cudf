#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2018-2023, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "page_decode.cuh"

#include <io/utilities/column_buffer.hpp>

#include <cudf/hashing/detail/murmurhash3_x86_32.cuh>

#include <rmm/exec_policy.hpp>
#include <thrust/reduce.h>

namespace cudf::io::parquet::detail {

namespace {

constexpr int decode_block_size = 128;
constexpr int rolling_buf_size  = decode_block_size * 2;

/**
 * @brief Output a string descriptor
 *
 * @param[in,out] s Page state input/output
 * @param[out] sb Page state buffer output
 * @param[in] src_pos Source position
 * @param[in] dstv Pointer to row output data (string descriptor or 32-bit hash)
 */
template <typename state_buf>
inline __device__ void gpuOutputString(volatile page_state_s* s,
                                       volatile state_buf* sb,
                                       int src_pos,
                                       void* dstv)
{
  auto [ptr, len] = gpuGetStringData(s, sb, src_pos);
  // make sure to only hash `BYTE_ARRAY` when specified with the output type size
  if (s->dtype_len == 4 and (s->col.data_type & 7) == BYTE_ARRAY) {
    // Output hash. This hash value is used if the option to convert strings to
    // categoricals is enabled. The seed value is chosen arbitrarily.
    uint32_t constexpr hash_seed = 33;
    cudf::string_view const sv{ptr, static_cast<size_type>(len)};
    *static_cast<uint32_t*>(dstv) =
      cudf::hashing::detail::MurmurHash3_x86_32<cudf::string_view>{hash_seed}(sv);
  } else {
    // Output string descriptor
    auto* dst   = static_cast<string_index_pair*>(dstv);
    dst->first  = ptr;
    dst->second = len;
  }
}

/**
 * @brief Output a boolean
 *
 * @param[out] sb Page state buffer output
 * @param[in] src_pos Source position
 * @param[in] dst Pointer to row output data
 */
template <typename state_buf>
inline __device__ void gpuOutputBoolean(volatile state_buf* sb, int src_pos, uint8_t* dst)
{
  *dst = sb->dict_idx[rolling_index<state_buf::dict_buf_size>(src_pos)];
}

/**
 * @brief Store a 32-bit data element
 *
 * @param[out] dst ptr to output
 * @param[in] src8 raw input bytes
 * @param[in] dict_pos byte position in dictionary
 * @param[in] dict_size size of dictionary
 */
inline __device__ void gpuStoreOutput(uint32_t* dst,
                                      uint8_t const* src8,
                                      uint32_t dict_pos,
                                      uint32_t dict_size)
{
  uint32_t bytebuf;
  unsigned int ofs = 3 & reinterpret_cast<size_t>(src8);
  src8 -= ofs;  // align to 32-bit boundary
  ofs <<= 3;    // bytes -> bits
  if (dict_pos < dict_size) {
    bytebuf = *reinterpret_cast<uint32_t const*>(src8 + dict_pos);
    if (ofs) {
      uint32_t bytebufnext = *reinterpret_cast<uint32_t const*>(src8 + dict_pos + 4);
      bytebuf              = __funnelshift_r(bytebuf, bytebufnext, ofs);
    }
  } else {
    bytebuf = 0;
  }
  *dst = bytebuf;
}

/**
 * @brief Store a 64-bit data element
 *
 * @param[out] dst ptr to output
 * @param[in] src8 raw input bytes
 * @param[in] dict_pos byte position in dictionary
 * @param[in] dict_size size of dictionary
 */
inline __device__ void gpuStoreOutput(uint2* dst,
                                      uint8_t const* src8,
                                      uint32_t dict_pos,
                                      uint32_t dict_size)
{
  uint2 v;
  unsigned int ofs = 3 & reinterpret_cast<size_t>(src8);
  src8 -= ofs;  // align to 32-bit boundary
  ofs <<= 3;    // bytes -> bits
  if (dict_pos < dict_size) {
    v.x = *reinterpret_cast<uint32_t const*>(src8 + dict_pos + 0);
    v.y = *reinterpret_cast<uint32_t const*>(src8 + dict_pos + 4);
    if (ofs) {
      uint32_t next = *reinterpret_cast<uint32_t const*>(src8 + dict_pos + 8);
      v.x           = __funnelshift_r(v.x, v.y, ofs);
      v.y           = __funnelshift_r(v.y, next, ofs);
    }
  } else {
    v.x = v.y = 0;
  }
  *dst = v;
}

/**
 * @brief Convert an INT96 Spark timestamp to 64-bit timestamp
 *
 * @param[in,out] s Page state input/output
 * @param[out] sb Page state buffer output
 * @param[in] src_pos Source position
 * @param[out] dst Pointer to row output data
 */
template <typename state_buf>
inline __device__ void gpuOutputInt96Timestamp(volatile page_state_s* s,
                                               volatile state_buf* sb,
                                               int src_pos,
                                               int64_t* dst)
{
  using cuda::std::chrono::duration_cast;

  uint8_t const* src8;
  uint32_t dict_pos, dict_size = s->dict_size, ofs;

  if (s->dict_base) {
    // Dictionary
    dict_pos =
      (s->dict_bits > 0) ? sb->dict_idx[rolling_index<state_buf::dict_buf_size>(src_pos)] : 0;
    src8 = s->dict_base;
  } else {
    // Plain
    dict_pos = src_pos;
    src8     = s->data_start;
  }
  dict_pos *= (uint32_t)s->dtype_len_in;
  ofs = 3 & reinterpret_cast<size_t>(src8);
  src8 -= ofs;  // align to 32-bit boundary
  ofs <<= 3;    // bytes -> bits

  if (dict_pos + 4 >= dict_size) {
    *dst = 0;
    return;
  }

  uint3 v;
  int64_t nanos, days;
  v.x = *reinterpret_cast<uint32_t const*>(src8 + dict_pos + 0);
  v.y = *reinterpret_cast<uint32_t const*>(src8 + dict_pos + 4);
  v.z = *reinterpret_cast<uint32_t const*>(src8 + dict_pos + 8);
  if (ofs) {
    uint32_t next = *reinterpret_cast<uint32_t const*>(src8 + dict_pos + 12);
    v.x           = __funnelshift_r(v.x, v.y, ofs);
    v.y           = __funnelshift_r(v.y, v.z, ofs);
    v.z           = __funnelshift_r(v.z, next, ofs);
  }
  nanos = v.y;
  nanos <<= 32;
  nanos |= v.x;
  // Convert from Julian day at noon to UTC seconds
  days = static_cast<int32_t>(v.z);
  cudf::duration_D d_d{
    days - 2440588};  // TBD: Should be noon instead of midnight, but this matches pyarrow

  *dst = [&]() {
    switch (s->col.ts_clock_rate) {
      case 1:  // seconds
        return duration_cast<duration_s>(d_d).count() +
               duration_cast<duration_s>(duration_ns{nanos}).count();
      case 1'000:  // milliseconds
        return duration_cast<duration_ms>(d_d).count() +
               duration_cast<duration_ms>(duration_ns{nanos}).count();
      case 1'000'000:  // microseconds
        return duration_cast<duration_us>(d_d).count() +
               duration_cast<duration_us>(duration_ns{nanos}).count();
      case 1'000'000'000:  // nanoseconds
      default: return duration_cast<cudf::duration_ns>(d_d).count() + nanos;
    }
  }();
}

/**
 * @brief Output a 64-bit timestamp
 *
 * @param[in,out] s Page state input/output
 * @param[out] sb Page state buffer output
 * @param[in] src_pos Source position
 * @param[in] dst Pointer to row output data
 */
template <typename state_buf>
inline __device__ void gpuOutputInt64Timestamp(volatile page_state_s* s,
                                               volatile state_buf* sb,
                                               int src_pos,
                                               int64_t* dst)
{
  uint8_t const* src8;
  uint32_t dict_pos, dict_size = s->dict_size, ofs;
  int64_t ts;

  if (s->dict_base) {
    // Dictionary
    dict_pos =
      (s->dict_bits > 0) ? sb->dict_idx[rolling_index<state_buf::dict_buf_size>(src_pos)] : 0;
    src8 = s->dict_base;
  } else {
    // Plain
    dict_pos = src_pos;
    src8     = s->data_start;
  }
  dict_pos *= (uint32_t)s->dtype_len_in;
  ofs = 3 & reinterpret_cast<size_t>(src8);
  src8 -= ofs;  // align to 32-bit boundary
  ofs <<= 3;    // bytes -> bits
  if (dict_pos + 4 < dict_size) {
    uint2 v;
    int64_t val;
    int32_t ts_scale;
    v.x = *reinterpret_cast<uint32_t const*>(src8 + dict_pos + 0);
    v.y = *reinterpret_cast<uint32_t const*>(src8 + dict_pos + 4);
    if (ofs) {
      uint32_t next = *reinterpret_cast<uint32_t const*>(src8 + dict_pos + 8);
      v.x           = __funnelshift_r(v.x, v.y, ofs);
      v.y           = __funnelshift_r(v.y, next, ofs);
    }
    val = v.y;
    val <<= 32;
    val |= v.x;
    // Output to desired clock rate
    ts_scale = s->ts_scale;
    if (ts_scale < 0) {
      // round towards negative infinity
      int sign = (val < 0);
      ts       = ((val + sign) / -ts_scale) + sign;
    } else {
      ts = val * ts_scale;
    }
  } else {
    ts = 0;
  }
  *dst = ts;
}

/**
 * @brief Output a byte array as int.
 *
 * @param[in] ptr Pointer to the byte array
 * @param[in] len Byte array length
 * @param[out] dst Pointer to row output data
 */
template <typename T>
__device__ void gpuOutputByteArrayAsInt(char const* ptr, int32_t len, T* dst)
{
  T unscaled = 0;
  for (auto i = 0; i < len; i++) {
    uint8_t v = ptr[i];
    unscaled  = (unscaled << 8) | v;
  }
  // Shift the unscaled value up and back down when it isn't all 8 bytes,
  // which sign extend the value for correctly representing negative numbers.
  unscaled <<= (sizeof(T) - len) * 8;
  unscaled >>= (sizeof(T) - len) * 8;
  *dst = unscaled;
}

/**
 * @brief Output a fixed-length byte array as int.
 *
 * @param[in,out] s Page state input/output
 * @param[out] sb Page state buffer output
 * @param[in] src_pos Source position
 * @param[in] dst Pointer to row output data
 */
template <typename T, typename state_buf>
__device__ void gpuOutputFixedLenByteArrayAsInt(volatile page_state_s* s,
                                                volatile state_buf* sb,
                                                int src_pos,
                                                T* dst)
{
  uint32_t const dtype_len_in = s->dtype_len_in;
  uint8_t const* data         = s->dict_base ? s->dict_base : s->data_start;
  uint32_t const pos =
    (s->dict_base
       ? ((s->dict_bits > 0) ? sb->dict_idx[rolling_index<state_buf::dict_buf_size>(src_pos)] : 0)
       : src_pos) *
    dtype_len_in;
  uint32_t const dict_size = s->dict_size;

  T unscaled = 0;
  for (unsigned int i = 0; i < dtype_len_in; i++) {
    uint32_t v = (pos + i < dict_size) ? data[pos + i] : 0;
    unscaled   = (unscaled << 8) | v;
  }
  // Shift the unscaled value up and back down when it isn't all 8 bytes,
  // which sign extend the value for correctly representing negative numbers.
  if (dtype_len_in < sizeof(T)) {
    unscaled <<= (sizeof(T) - dtype_len_in) * 8;
    unscaled >>= (sizeof(T) - dtype_len_in) * 8;
  }
  *dst = unscaled;
}

/**
 * @brief Output a small fixed-length value
 *
 * @param[in,out] s Page state input/output
 * @param[out] sb Page state buffer output
 * @param[in] src_pos Source position
 * @param[in] dst Pointer to row output data
 */
template <typename T, typename state_buf>
inline __device__ void gpuOutputFast(volatile page_state_s* s,
                                     volatile state_buf* sb,
                                     int src_pos,
                                     T* dst)
{
  uint8_t const* dict;
  uint32_t dict_pos, dict_size = s->dict_size;

  if (s->dict_base) {
    // Dictionary
    dict_pos =
      (s->dict_bits > 0) ? sb->dict_idx[rolling_index<state_buf::dict_buf_size>(src_pos)] : 0;
    dict = s->dict_base;
  } else {
    // Plain
    dict_pos = src_pos;
    dict     = s->data_start;
  }
  dict_pos *= (uint32_t)s->dtype_len_in;
  gpuStoreOutput(dst, dict, dict_pos, dict_size);
}

/**
 * @brief Output a N-byte value
 *
 * @param[in,out] s Page state input/output
 * @param[out] sb Page state buffer output
 * @param[in] src_pos Source position
 * @param[in] dst8 Pointer to row output data
 * @param[in] len Length of element
 */
template <typename state_buf>
static __device__ void gpuOutputGeneric(
  volatile page_state_s* s, volatile state_buf* sb, int src_pos, uint8_t* dst8, int len)
{
  uint8_t const* dict;
  uint32_t dict_pos, dict_size = s->dict_size;

  if (s->dict_base) {
    // Dictionary
    dict_pos =
      (s->dict_bits > 0) ? sb->dict_idx[rolling_index<state_buf::dict_buf_size>(src_pos)] : 0;
    dict = s->dict_base;
  } else {
    // Plain
    dict_pos = src_pos;
    dict     = s->data_start;
  }
  dict_pos *= (uint32_t)s->dtype_len_in;
  if (len & 3) {
    // Generic slow path
    for (unsigned int i = 0; i < len; i++) {
      dst8[i] = (dict_pos + i < dict_size) ? dict[dict_pos + i] : 0;
    }
  } else {
    // Copy 4 bytes at a time
    uint8_t const* src8 = dict;
    unsigned int ofs    = 3 & reinterpret_cast<size_t>(src8);
    src8 -= ofs;  // align to 32-bit boundary
    ofs <<= 3;    // bytes -> bits
    for (unsigned int i = 0; i < len; i += 4) {
      uint32_t bytebuf;
      if (dict_pos < dict_size) {
        bytebuf = *reinterpret_cast<uint32_t const*>(src8 + dict_pos);
        if (ofs) {
          uint32_t bytebufnext = *reinterpret_cast<uint32_t const*>(src8 + dict_pos + 4);
          bytebuf              = __funnelshift_r(bytebuf, bytebufnext, ofs);
        }
      } else {
        bytebuf = 0;
      }
      dict_pos += 4;
      *reinterpret_cast<uint32_t*>(dst8 + i) = bytebuf;
    }
  }
}

/**
 * @brief Kernel for computing the column data stored in the pages
 *
 * This function will write the page data and the page data's validity to the
 * output specified in the page's column chunk. If necessary, additional
 * conversion will be performed to translate from the Parquet datatype to
 * desired output datatype (ex. 32-bit to 16-bit, string to hash).
 *
 * @param pages List of pages
 * @param chunks List of column chunks
 * @param min_row Row index to start reading at
 * @param num_rows Maximum number of rows to read
 * @param error_code Error code to set if an error is encountered
 */
template <int lvl_buf_size, typename level_t>
__global__ void __launch_bounds__(decode_block_size)
  gpuDecodePageData(PageInfo* pages,
                    device_span<ColumnChunkDesc const> chunks,
                    size_t min_row,
                    size_t num_rows,
                    int32_t* error_code)
{
  __shared__ __align__(16) page_state_s state_g;
  __shared__ __align__(16)
    page_state_buffers_s<rolling_buf_size, rolling_buf_size, rolling_buf_size>
      state_buffers;

  page_state_s* const s = &state_g;
  auto* const sb        = &state_buffers;
  int page_idx          = blockIdx.x;
  int t                 = threadIdx.x;
  int out_thread0;
  [[maybe_unused]] null_count_back_copier _{s, t};

  if (!setupLocalPageInfo(s,
                          &pages[page_idx],
                          chunks,
                          min_row,
                          num_rows,
                          mask_filter{decode_kernel_mask::GENERAL},
                          true)) {
    return;
  }

  bool const has_repetition = s->col.max_level[level_type::REPETITION] > 0;

  if (s->dict_base) {
    out_thread0 = (s->dict_bits > 0) ? 64 : 32;
  } else {
    switch (s->col.data_type & 7) {
      case BOOLEAN: [[fallthrough]];
      case BYTE_ARRAY: [[fallthrough]];
      case FIXED_LEN_BYTE_ARRAY: out_thread0 = 64; break;
      default: out_thread0 = 32;
    }
  }

  PageNestingDecodeInfo* nesting_info_base = s->nesting_info;

  __shared__ level_t rep[rolling_buf_size];  // circular buffer of repetition level values
  __shared__ level_t def[rolling_buf_size];  // circular buffer of definition level values

  // skipped_leaf_values will always be 0 for flat hierarchies.
  uint32_t skipped_leaf_values = s->page.skipped_leaf_values;
  while (s->error == 0 &&
         (s->input_value_count < s->num_input_values || s->src_pos < s->nz_count)) {
    int target_pos;
    int src_pos = s->src_pos;

    if (t < out_thread0) {
      target_pos = min(src_pos + 2 * (decode_block_size - out_thread0),
                       s->nz_count + (decode_block_size - out_thread0));
    } else {
      target_pos = min(s->nz_count, src_pos + decode_block_size - out_thread0);
      if (out_thread0 > 32) { target_pos = min(target_pos, s->dict_pos); }
    }
    // TODO(ets): see if this sync can be removed
    __syncthreads();
    if (t < 32) {
      // decode repetition and definition levels.
      // - update validity vectors
      // - updates offsets (for nested columns)
      // - produces non-NULL value indices in s->nz_idx for subsequent decoding
      gpuDecodeLevels<lvl_buf_size, level_t>(s, sb, target_pos, rep, def, t);
    } else if (t < out_thread0) {
      // skipped_leaf_values will always be 0 for flat hierarchies.
      uint32_t src_target_pos = target_pos + skipped_leaf_values;

      // WARP1: Decode dictionary indices, booleans or string positions
      if (s->dict_base) {
        src_target_pos = gpuDecodeDictionaryIndices<false>(s, sb, src_target_pos, t & 0x1f).first;
      } else if ((s->col.data_type & 7) == BOOLEAN) {
        src_target_pos = gpuDecodeRleBooleans(s, sb, src_target_pos, t & 0x1f);
      } else if ((s->col.data_type & 7) == BYTE_ARRAY or
                 (s->col.data_type & 7) == FIXED_LEN_BYTE_ARRAY) {
        gpuInitStringDescriptors<false>(s, sb, src_target_pos, t & 0x1f);
      }
      if (t == 32) { *(volatile int32_t*)&s->dict_pos = src_target_pos; }
    } else {
      // WARP1..WARP3: Decode values
      int const dtype = s->col.data_type & 7;
      src_pos += t - out_thread0;

      // the position in the output column/buffer
      int dst_pos = sb->nz_idx[rolling_index<rolling_buf_size>(src_pos)];

      // for the flat hierarchy case we will be reading from the beginning of the value stream,
      // regardless of the value of first_row. so adjust our destination offset accordingly.
      // example:
      // - user has passed skip_rows = 2, so our first_row to output is 2
      // - the row values we get from nz_idx will be
      //   0, 1, 2, 3, 4 ....
      // - by shifting these values by first_row, the sequence becomes
      //   -1, -2, 0, 1, 2 ...
      // - so we will end up ignoring the first two input rows, and input rows 2..n will
      //   get written to the output starting at position 0.
      //
      if (!has_repetition) { dst_pos -= s->first_row; }

      // target_pos will always be properly bounded by num_rows, but dst_pos may be negative (values
      // before first_row) in the flat hierarchy case.
      if (src_pos < target_pos && dst_pos >= 0) {
        // src_pos represents the logical row position we want to read from. But in the case of
        // nested hierarchies, there is no 1:1 mapping of rows to values.  So our true read position
        // has to take into account the # of values we have to skip in the page to get to the
        // desired logical row.  For flat hierarchies, skipped_leaf_values will always be 0.
        uint32_t val_src_pos = src_pos + skipped_leaf_values;

        // nesting level that is storing actual leaf values
        int leaf_level_index = s->col.max_nesting_depth - 1;

        uint32_t dtype_len = s->dtype_len;
        void* dst =
          nesting_info_base[leaf_level_index].data_out + static_cast<size_t>(dst_pos) * dtype_len;
        if (dtype == BYTE_ARRAY) {
          if (s->col.converted_type == DECIMAL) {
            auto const [ptr, len]        = gpuGetStringData(s, sb, val_src_pos);
            auto const decimal_precision = s->col.decimal_precision;
            if (decimal_precision <= MAX_DECIMAL32_PRECISION) {
              gpuOutputByteArrayAsInt(ptr, len, static_cast<int32_t*>(dst));
            } else if (decimal_precision <= MAX_DECIMAL64_PRECISION) {
              gpuOutputByteArrayAsInt(ptr, len, static_cast<int64_t*>(dst));
            } else {
              gpuOutputByteArrayAsInt(ptr, len, static_cast<__int128_t*>(dst));
            }
          } else {
            gpuOutputString(s, sb, val_src_pos, dst);
          }
        } else if (dtype == BOOLEAN) {
          gpuOutputBoolean(sb, val_src_pos, static_cast<uint8_t*>(dst));
        } else if (s->col.converted_type == DECIMAL) {
          switch (dtype) {
            case INT32: gpuOutputFast(s, sb, val_src_pos, static_cast<uint32_t*>(dst)); break;
            case INT64: gpuOutputFast(s, sb, val_src_pos, static_cast<uint2*>(dst)); break;
            default:
              if (s->dtype_len_in <= sizeof(int32_t)) {
                gpuOutputFixedLenByteArrayAsInt(s, sb, val_src_pos, static_cast<int32_t*>(dst));
              } else if (s->dtype_len_in <= sizeof(int64_t)) {
                gpuOutputFixedLenByteArrayAsInt(s, sb, val_src_pos, static_cast<int64_t*>(dst));
              } else {
                gpuOutputFixedLenByteArrayAsInt(s, sb, val_src_pos, static_cast<__int128_t*>(dst));
              }
              break;
          }
        } else if (dtype == FIXED_LEN_BYTE_ARRAY) {
          gpuOutputString(s, sb, val_src_pos, dst);
        } else if (dtype == INT96) {
          gpuOutputInt96Timestamp(s, sb, val_src_pos, static_cast<int64_t*>(dst));
        } else if (dtype_len == 8) {
          if (s->dtype_len_in == 4) {
            // Reading INT32 TIME_MILLIS into 64-bit DURATION_MILLISECONDS
            // TIME_MILLIS is the only duration type stored as int32:
            // https://github.com/apache/parquet-format/blob/master/LogicalTypes.md#deprecated-time-convertedtype
            gpuOutputFast(s, sb, val_src_pos, static_cast<uint32_t*>(dst));
          } else if (s->ts_scale) {
            gpuOutputInt64Timestamp(s, sb, val_src_pos, static_cast<int64_t*>(dst));
          } else {
            gpuOutputFast(s, sb, val_src_pos, static_cast<uint2*>(dst));
          }
        } else if (dtype_len == 4) {
          gpuOutputFast(s, sb, val_src_pos, static_cast<uint32_t*>(dst));
        } else {
          gpuOutputGeneric(s, sb, val_src_pos, static_cast<uint8_t*>(dst), dtype_len);
        }
      }

      if (t == out_thread0) { *(volatile int32_t*)&s->src_pos = target_pos; }
    }
    __syncthreads();
  }
  if (t == 0 and s->error != 0) { set_error(s->error, error_code); }
}

struct mask_tform {
  __device__ uint32_t operator()(PageInfo const& p) { return static_cast<uint32_t>(p.kernel_mask); }
};

}  // anonymous namespace

uint32_t GetAggregatedDecodeKernelMask(cudf::detail::hostdevice_vector<PageInfo>& pages,
                                       rmm::cuda_stream_view stream)
{
  // determine which kernels to invoke
  auto mask_iter = thrust::make_transform_iterator(pages.d_begin(), mask_tform{});
  return thrust::reduce(
    rmm::exec_policy(stream), mask_iter, mask_iter + pages.size(), 0U, thrust::bit_or<uint32_t>{});
}

/**
 * @copydoc cudf::io::parquet::detail::DecodePageData
 */
void __host__ DecodePageData(cudf::detail::hostdevice_vector<PageInfo>& pages,
                             cudf::detail::hostdevice_vector<ColumnChunkDesc> const& chunks,
                             size_t num_rows,
                             size_t min_row,
                             int level_type_size,
                             int32_t* error_code,
                             rmm::cuda_stream_view stream)
{
  CUDF_EXPECTS(pages.size() > 0, "There is no page to decode");

  dim3 dim_block(decode_block_size, 1);
  dim3 dim_grid(pages.size(), 1);  // 1 threadblock per page

  if (level_type_size == 1) {
    gpuDecodePageData<rolling_buf_size, uint8_t><<<dim_grid, dim_block, 0, stream.value()>>>(
      pages.device_ptr(), chunks, min_row, num_rows, error_code);
  } else {
    gpuDecodePageData<rolling_buf_size, uint16_t><<<dim_grid, dim_block, 0, stream.value()>>>(
      pages.device_ptr(), chunks, min_row, num_rows, error_code);
  }
}

}  // namespace cudf::io::parquet::detail
