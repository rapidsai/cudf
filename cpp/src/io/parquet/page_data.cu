#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2018, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "parquet_gpu.h"
#include <io/utilities/block_utils.cuh>

#define LOG2_NTHREADS   (5+2)
#define NTHREADS        (1 << LOG2_NTHREADS)
#define NZ_BFRSZ        (NTHREADS*2)


inline __device__ uint32_t rotl32(uint32_t x, uint32_t r)
{
    return __funnelshift_l(x, x, r);    // (x << r) | (x >> (32 - r));
};

namespace cudf {
namespace io {
namespace parquet {
namespace gpu {

struct page_state_s {
    const uint8_t *lvl_start[2];  // [def,rep]
    const uint8_t *data_start;
    const uint8_t *data_end;
    uint32_t *valid_map;
    const uint8_t *dict_base;       // ptr to dictionary page data
    int32_t dict_size;              // size of dictionary data
    uint8_t *data_out;
    int32_t valid_map_offset;       // offset in valid_map, in bits
    uint32_t out_valid;
    uint32_t out_valid_mask;
    int32_t first_row;              // First row in page to output
    int32_t num_rows;               // Rows in page to decode (including rows to be skipped)
    int32_t dtype_len;              // Output data type length
    int32_t dtype_len_in;           // Can be larger than dtype_len if truncating 32-bit into 8-bit
    int32_t dict_bits;              // # of bits to store dictionary indices
    uint32_t dict_run;
    int32_t dict_val;
    uint32_t initial_rle_run[2];    // [def,rep]
    int32_t initial_rle_value[2];   // [def,rep]
    int32_t error;
    PageInfo page;
    ColumnChunkDesc col;
    int32_t value_count;            // number of values decoded (including NULLs)
    int32_t num_values;             // max number of values to decode
    int32_t nz_count;               // number of valid entries in nz_idx (write position in circular buffer)
    int32_t dict_pos;               // write position of dictionary indices
    int32_t out_pos;                // read position of final output
    int32_t int64_nanoscale;        // if non-zero, convert timestamp to nano
    uint32_t nz_idx[NZ_BFRSZ];      // circular buffer of non-null row positions
    uint32_t dict_idx[NZ_BFRSZ];    // Dictionary index, boolean, or string offset values
    uint32_t str_len[NZ_BFRSZ];     // String length for plain encoding of strings
};


/**---------------------------------------------------------------------------*
* @brief Computes a 32-bit hash when given a byte stream and range.
*
* MurmurHash3_32 implementation from
* https://github.com/aappleby/smhasher/blob/master/src/MurmurHash3.cpp
*
* MurmurHash3 was written by Austin Appleby, and is placed in the public
* domain. The author hereby disclaims copyright to this source code.
*
* @param[in] key The input data to hash
* @param[in] len The length of the input data
* @param[in] seed An initialization value
*
* @return The hash value
*---------------------------------------------------------------------------**/
__device__ uint32_t device_str2hash32(const char* key, size_t len, uint32_t seed = 33)
{
    const uint8_t *p = reinterpret_cast<const uint8_t *>(key);
    uint32_t h1 = seed, k1;
    const uint32_t c1 = 0xcc9e2d51;
    const uint32_t c2 = 0x1b873593;
    int l = len;
    // body
    while (l >= 4)
    {
        k1 = p[0] | (p[1] << 8) | (p[2] << 16) | (p[3] << 24);
        k1 *= c1;
        k1 = rotl32(k1, 15);
        k1 *= c2;
        h1 ^= k1;
        h1 = rotl32(h1, 13);
        h1 = h1 * 5 + 0xe6546b64;
        p += 4;
        l -= 4;
    }
    // tail
    k1 = 0;
    switch (l)
    {
    case 3:
        k1 ^= p[2] << 16;
    case 2:
        k1 ^= p[1] << 8;
    case 1:
        k1 ^= p[0];
        k1 *= c1;
        k1 = rotl32(k1, 15);
        k1 *= c2;
        h1 ^= k1;
    }
    // finalization
    h1 ^= len;
    h1 ^= h1 >> 16;
    h1 *= 0x85ebca6b;
    h1 ^= h1 >> 13;
    h1 *= 0xc2b2ae35;
    h1 ^= h1 >> 16;
    return h1;
}

/**
 * @brief Read a 32-bit varint integer
 *
 * @param[in,out] cur The current data position, updated after the read
 * @param[in] end The end data position
 *
 * @return The 32-bit value read
 **/
inline __device__ uint32_t get_vlq32(const uint8_t *&cur, const uint8_t *end)
{
    uint32_t v = *cur++;
    if (v >= 0x80 && cur < end)
    {
        v = (v & 0x7f) | ((*cur++) << 7);
        if (v >= (0x80 << 7) && cur < end)
        {
            v = (v & ((0x7f << 7) | 0x7f)) | ((*cur++) << 14);
            if (v >= (0x80 << 14) && cur < end)
            {
                v = (v & ((0x7f << 14) | (0x7f << 7) | 0x7f)) | ((*cur++) << 21);
                if (v >= (0x80 << 21) && cur < end)
                {
                    v = (v & ((0x7f << 21) | (0x7f << 14) | (0x7f << 7) | 0x7f)) | ((*cur++) << 28);
                }
            }
        }
    }
    return v;
}

/**
 * @brief Parse the beginning of the level section (definition or repetition),
 * initializes the initial RLE run & value, and returns the section length
 *
 * @param[in,out] s The page state
 * @param[in] cur The current data position
 * @param[in] end The end of the data
 * @param[in] encoding The encoding type
 * @param[in] level_bits The bits required
 * @param[in] idx The index into the output section
 **/
__device__ uint32_t InitLevelSection(page_state_s *s, const uint8_t *cur, const uint8_t *end, int encoding, int level_bits, int idx)
{
    int32_t len;
    if (level_bits == 0)
    {
        len = 0;
        s->initial_rle_run[idx] = s->page.num_values * 2; // repeated value
        s->initial_rle_value[idx] = 0;
        s->lvl_start[idx] = cur;
    }
    else if (encoding == RLE)
    {
        if (cur + 4 < end)
        {
            uint32_t run;
            len = 4 + (cur[0]) + (cur[1] << 8) + (cur[2] << 16) + (cur[3] << 24);
            cur += 4;
            run = get_vlq32(cur, end);
            s->initial_rle_run[idx] = run;
            if (!(run & 1))
            {
                int v = (cur < end) ? cur[0] : 0;
                cur++;
                if (level_bits > 8)
                {
                    v |= ((cur < end) ? cur[0] : 0) << 8;
                    cur++;
                }
                s->initial_rle_value[idx] = v;
            }
            s->lvl_start[idx] = cur;
            if (cur > end)
            {
                s->error = 2;
            }
        }
        else
        {
            len = 0;
            s->error = 2;
        }
    }
    else if (encoding == BIT_PACKED)
    {
        len = (s->page.num_values * level_bits + 7) >> 3;
        s->initial_rle_run[idx] = ((s->page.num_values + 7) >> 3) * 2 + 1; // literal run
        s->initial_rle_value[idx] = 0;
        s->lvl_start[idx] = cur;
    }
    else
    {
        s->error = 3;
        len = 0;
    }
    return (uint32_t)len;
}

/**
 * @brief Decode definition and repetition levels and outputs row indices
 *
 * @param[in,out] s Page state input/output
 * @param[in] t target_count Target count of non-NULL values on output
 * @param[in] t Warp0 thread ID (0..31)
 **/
__device__ void gpuDecodeLevels(page_state_s *s, int32_t target_count, int t)
{
    const uint8_t *cur_def = s->lvl_start[0];
    const uint8_t *end = s->data_start;
    uint32_t *valid_map = s->valid_map;
    uint32_t valid_map_offset = s->valid_map_offset;
    uint32_t out_valid = s->out_valid, out_valid_mask = s->out_valid_mask;
    int32_t first_row = s->first_row;
    uint32_t def_run = s->initial_rle_run[0];
    int32_t def_val = s->initial_rle_value[0];
    int def_bits = s->col.def_level_bits;
    int max_def_level = s->col.max_def_level;
    int32_t num_values = s->num_values;
    int32_t value_count = s->value_count;   // Row offset of next value
    int32_t coded_count = s->nz_count;      // Count of non-null values
    while (coded_count < target_count && value_count < num_values)
    {
        int batch_len, is_valid;
        uint32_t valid_mask;
        if (def_run <= 1)
        {
            // Get a new run symbol from the byte stream
            int sym_len = 0;
            if (!t)
            {
                const uint8_t *cur = cur_def;
                if (cur < end)
                {
                    def_run = get_vlq32(cur, end);
                }
                if (!(def_run & 1))
                {
                    if (cur < end)
                        def_val = cur[0];
                    cur++;
                    if (def_bits > 8)
                    {
                        if (cur < end)
                            def_val |= cur[0] << 8;
                        cur++;
                    }
                }
                if (cur > end || def_run <= 1)
                {
                    s->error = 0x10;
                }
                sym_len = (int32_t)(cur - cur_def);
                __threadfence_block();
            }
            sym_len = SHFL0(sym_len);
            def_val = SHFL0(def_val);
            def_run = SHFL0(def_run);
            cur_def += sym_len;
        }
        if (s->error)
        {
            break;
        }
        batch_len = min(num_values - value_count, 32);
        if (def_run & 1)
        {
            // Literal run
            int batch_len8;
            batch_len = min(batch_len, (def_run>>1)*8);
            batch_len8 = (batch_len + 7) >> 3;
            if (t < batch_len)
            {
                int bitpos = t * def_bits;
                const uint8_t *cur = cur_def + (bitpos >> 3);
                bitpos &= 7;
                if (cur < end)
                    def_val = cur[0];
                cur++;
                if (def_bits > 8 - bitpos && cur < end)
                {
                    def_val |= cur[0] << 8;
                    cur++;
                    if (def_bits > 16 - bitpos && cur < end)
                        def_val |= cur[0] << 16;
                }
                def_val = (def_val >> bitpos) & ((1 << def_bits) - 1);
            }
            def_run -= batch_len8 * 2;
            cur_def += batch_len8 * def_bits;
        }
        else
        {
            // Repeated value
            batch_len = min(batch_len, def_run >> 1);
            def_run -= batch_len * 2;
        }
        is_valid = (t < batch_len && def_val >= max_def_level);
        valid_mask = BALLOT(is_valid);
        if (valid_mask)
        {
            if (is_valid)
            {
                int idx = coded_count + __popc(valid_mask & ((1 << t) - 1));
                int ofs = value_count + t - first_row;
                s->nz_idx[idx & (NZ_BFRSZ - 1)] = ofs;
            }
            coded_count += __popc(valid_mask);
        }
        value_count += batch_len;
        if (!t && valid_map)
        {
            // If needed, adjust batch length to eliminate rows before the first row
            if (value_count < first_row + batch_len)
            {
                if (value_count > first_row)
                {
                    // At least some values are above the first row
                    unsigned int skip_cnt = batch_len - (value_count - first_row);
                    valid_mask >>= skip_cnt;
                    batch_len -= skip_cnt;
                }
                else // All values are below the first row
                {
                    batch_len = 0;
                    valid_mask = 0;
                }
            }
            out_valid |= valid_mask << valid_map_offset;
            valid_map_offset += batch_len;
            if (valid_map_offset >= 32)
            {
                if (out_valid_mask == ~0) // Safe to output all 32 bits are within the current page
                {
                    *valid_map = out_valid;
                }
                else // Special case for the first valid row, which may not start on a 32-bit boundary (only setting some of the bits)
                {
                    atomicAnd(valid_map, ~out_valid_mask);
                    atomicOr(valid_map, out_valid);
                }
                s->page.valid_count += __popc(out_valid);
                valid_map_offset &= 0x1f;
                out_valid = (valid_map_offset > 0) ? valid_mask >> (unsigned int)(batch_len - valid_map_offset) : 0;
                out_valid_mask = ~0;
                valid_map++;
            }
            __threadfence_block();
        }
    }
    if (!t)
    {
        s->lvl_start[0] = cur_def;
        s->initial_rle_run[0] = def_run;
        s->initial_rle_value[0] = def_val;
        if (value_count >= num_values && valid_map && valid_map_offset != 0)
        {
            // Store the remaining valid bits at the end of the page
            out_valid_mask &= (1 << valid_map_offset) - 1;
            out_valid &= out_valid_mask;
            s->page.valid_count += __popc(out_valid);
            atomicAnd(valid_map, ~out_valid_mask);
            atomicOr(valid_map, out_valid);
            out_valid_mask = 0;
        }
        s->valid_map_offset = valid_map_offset;
        s->out_valid_mask = out_valid_mask;
        s->out_valid = out_valid;
        s->valid_map = valid_map;
        s->value_count = value_count;
        s->nz_count = coded_count;
    }
}

/**
 * @brief Performs RLE decoding of dictionary indexes
 *
 * @param[in,out] s Page state input/output
 * @param[in] target_pos Target index position in dict_idx buffer (may exceed this value by up to 31)
 * @param[in] t Warp1 thread ID (0..31)
 *
 * @return The new output position
 **/
__device__ int gpuDecodeDictionaryIndices(volatile page_state_s *s, int target_pos, int t)
{
    const uint8_t *end = s->data_end;
    int dict_bits = s->dict_bits;
    int pos = s->dict_pos;

    while (pos < target_pos)
    {
        int is_literal, batch_len;
        if (!t)
        {
            uint32_t run = s->dict_run;
            const uint8_t *cur = s->data_start;
            if (run <= 1)
            {
                run = (cur < end) ? get_vlq32(cur, end) : 0;
                if (!(run & 1))
                {
                    // Repeated value
                    int bytecnt = (dict_bits + 7) >> 3;
                    if (cur + bytecnt <= end)
                    {
                        int32_t run_val = cur[0];
                        if (bytecnt > 1)
                        {
                            run_val |= cur[1] << 8;
                            if (bytecnt > 2)
                            {
                                run_val |= cur[2] << 16;
                                if (bytecnt > 3)
                                {
                                    run_val |= cur[3] << 24;
                                }
                            }
                        }
                        s->dict_val = run_val & ((1 << dict_bits) - 1);
                    }
                    cur += bytecnt;
                }
            }
            if (run & 1)
            {
                // Literal batch: must output a multiple of 8, except for the last batch
                int batch_len_div8;
                batch_len = max(min(32, (int)(run >> 1) * 8), 1);
                batch_len_div8 = (batch_len + 7) >> 3;
                run -= batch_len_div8 * 2;
                cur += batch_len_div8 * dict_bits;
            }
            else
            {
                batch_len = max(min(32, (int)(run >> 1)), 1);
                run -= batch_len * 2;
            }
            s->dict_run = run;
            s->data_start = cur;
            is_literal = run & 1;
            __threadfence_block();
        }
        SYNCWARP();
        is_literal = SHFL0(is_literal);
        batch_len = SHFL0(batch_len);
        if (t < batch_len)
        {
            int dict_idx = s->dict_val;
            if (is_literal)
            {
                int32_t ofs = (t - ((batch_len + 7) & ~7)) * dict_bits;
                const uint8_t *p = s->data_start + (ofs >> 3);
                ofs &= 7;
                if (p < end)
                {
                    uint32_t c = 8 - ofs;
                    dict_idx = (*p++) >> ofs;
                    if (c < dict_bits && p < end)
                    {
                        dict_idx |= (*p++) << c;
                        c += 8;
                        if (c < dict_bits && p < end)
                        {
                            dict_idx |= (*p++) << c;
                            c += 8;
                            if (c < dict_bits && p < end)
                            {
                                dict_idx |= (*p++) << c;
                            }
                        }
                    }
                    dict_idx &= (1 << dict_bits) - 1;
                }
            }
            s->dict_idx[(pos + t) & (NZ_BFRSZ - 1)] = dict_idx;
        }
        pos += batch_len;
    }
    return pos;
}

/**
 * @brief Performs RLE decoding of dictionary indexes, for when dict_size=1
 *
 * @param[in,out] s Page state input/output
 * @param[in] target_pos Target write position
 * @param[in] t Thread ID
 *
 * @return The new output position
 **/
__device__ int gpuDecodeRleBooleans(volatile page_state_s *s, int target_pos, int t)
{
    const uint8_t *end = s->data_end;
    int pos = s->dict_pos;

    while (pos < target_pos)
    {
        int is_literal, batch_len;
        if (!t)
        {
            uint32_t run = s->dict_run;
            const uint8_t *cur = s->data_start;
            if (run <= 1)
            {
                run = (cur < end) ? get_vlq32(cur, end) : 0;
                if (!(run & 1))
                {
                    // Repeated value
                    s->dict_val = (cur < end) ? cur[0] & 1 : 0;
                    cur++;
                }
            }
            if (run & 1)
            {
                // Literal batch: must output a multiple of 8, except for the last batch
                int batch_len_div8;
                batch_len = max(min(32, (int)(run >> 1) * 8), 1);
                if (batch_len >= 8)
                {
                    batch_len &= ~7;
                }
                batch_len_div8 = (batch_len + 7) >> 3;
                run -= batch_len_div8 * 2;
                cur += batch_len_div8;
            }
            else
            {
                batch_len = max(min(32, (int)(run >> 1)), 1);
                run -= batch_len * 2;
            }
            s->dict_run = run;
            s->data_start = cur;
            is_literal = run & 1;
            __threadfence_block();
        }
        SYNCWARP();
        is_literal = SHFL0(is_literal);
        batch_len = SHFL0(batch_len);
        if (t < batch_len)
        {
            int dict_idx;
            if (is_literal)
            {
                int32_t ofs = t - ((batch_len + 7) & ~7);
                const uint8_t *p = s->data_start + (ofs >> 3);
                dict_idx = (p < end) ? (p[0] >> (ofs & 7u)) & 1 : 0;
            }
            else
            {
                dict_idx = s->dict_val;
            }
            s->dict_idx[(pos + t) & (NZ_BFRSZ - 1)] = dict_idx;
        }
        pos += batch_len;
    }
    return pos;
}

/**
 * @brief Parses the length and position of strings
 *
 * @param[in,out] s Page state input/output
 * @param[in] target_pos Target output position
 * @param[in] t Thread ID
 *
 * @return The new output position
 **/
__device__ void gpuInitStringDescriptors(volatile page_state_s *s, int target_pos, int t)
{
    int pos = s->dict_pos;
    // This step is purely serial
    if (!t)
    {
        const uint8_t *cur = s->data_start;
        int dict_size = s->dict_size;
        int k = s->dict_val;

        while (pos < target_pos)
        {
            int len;
            if (k + 4 <= dict_size)
            {
                len = (cur[k]) | (cur[k+1] << 8) | (cur[k+2] << 16) | (cur[k+3] << 24);
                k += 4;
                if (k + len > dict_size)
                {
                    len = 0;
                }
            }
            else
            {
                len = 0;
            }
            s->dict_idx[pos & (NZ_BFRSZ - 1)] = k;
            s->str_len[pos & (NZ_BFRSZ - 1)] = len;
            k += len;
            pos++;
        }
        s->dict_val = k;
        __threadfence_block();
    }
}


/**
 * @brief Output a string descriptor
 *
 * @param[in,out] s Page state input/output
 * @param[in] src_pos Source position
 * @param[in] dstv Pointer to row output data (string descriptor or 32-bit hash)
 **/
inline __device__ void gpuOutputString(volatile page_state_s *s, int src_pos, void *dstv)
{
    const char *ptr = NULL;
    size_t len = 0;

    if (s->dict_base)
    {
        // String dictionary
        uint32_t dict_pos = (s->dict_bits > 0) ? s->dict_idx[src_pos & (NZ_BFRSZ - 1)] * sizeof(nvstrdesc_s) : 0;
        if (dict_pos < (uint32_t)s->dict_size)
        {
            const nvstrdesc_s *src = reinterpret_cast<const nvstrdesc_s *>(s->dict_base + dict_pos);
            ptr = src->ptr;
            len = src->count;
        }
    }
    else
    {
        // Plain encoding
        uint32_t dict_pos = s->dict_idx[src_pos & (NZ_BFRSZ - 1)];
        if (dict_pos < (uint32_t)s->dict_size)
        {
            ptr = reinterpret_cast<const char *>(s->data_start + dict_pos);
            len = s->str_len[src_pos & (NZ_BFRSZ - 1)];
        }
    }
    if (s->dtype_len == 4)
    {
        // Output hash
        *reinterpret_cast<uint32_t *>(dstv) = device_str2hash32(ptr, len);
    }
    else
    {
        // Output string descriptor
        nvstrdesc_s *dst = reinterpret_cast<nvstrdesc_s *>(dstv);
        dst->ptr = ptr;
        dst->count = len;
    }
}


/**
 * @brief Output a boolean
 *
 * @param[in,out] s Page state input/output
 * @param[in] src_pos Source position
 * @param[in] dst Pointer to row output data
 **/
inline __device__ void gpuOutputBoolean(volatile page_state_s *s, int src_pos, uint8_t *dst)
{
    *dst = s->dict_idx[src_pos & (NZ_BFRSZ - 1)];
}


/**
 * @brief Store a 32-bit data element
 *
 * @param[out] dst ptr to output
 * @param[in] src8 raw input bytes
 * @param[in] dict_pos byte position in dictionary
 * @param[in] dict_size size of dictionary
 **/
inline __device__ void gpuStoreOutput(uint32_t *dst, const uint8_t *src8, uint32_t dict_pos, uint32_t dict_size)
{
    uint32_t bytebuf;
    unsigned int ofs = 3 & reinterpret_cast<size_t>(src8);
    src8 -= ofs;    // align to 32-bit boundary
    ofs <<= 3;      // bytes -> bits
    if (dict_pos < dict_size)
    {
        bytebuf = *(const uint32_t *)(src8 + dict_pos);
        if (ofs)
        {
            uint32_t bytebufnext = *(const uint32_t *)(src8 + dict_pos + 4);
            bytebuf = __funnelshift_r(bytebuf, bytebufnext, ofs);
        }
    }
    else
    {
        bytebuf = 0;
    }
    *dst = bytebuf;
}


/**
 * @brief Store a 64-bit data element
 *
 * @param[out] dst ptr to output
 * @param[in] src8 raw input bytes
 * @param[in] dict_pos byte position in dictionary
 * @param[in] dict_size size of dictionary
 **/
inline __device__ void gpuStoreOutput(uint2 *dst, const uint8_t *src8, uint32_t dict_pos, uint32_t dict_size)
{
    uint2 v;
    unsigned int ofs = 3 & reinterpret_cast<size_t>(src8);
    src8 -= ofs;    // align to 32-bit boundary
    ofs <<= 3;      // bytes -> bits
    if (dict_pos < dict_size)
    {
        v.x = *(const uint32_t *)(src8 + dict_pos + 0);
        v.y = *(const uint32_t *)(src8 + dict_pos + 4);
        if (ofs)
        {
            uint32_t next = *(const uint32_t *)(src8 + dict_pos + 8);
            v.x = __funnelshift_r(v.x, v.y, ofs);
            v.y = __funnelshift_r(v.y, next, ofs);
        }
    }
    else
    {
        v.x = v.y = 0;
    }
    *dst = v;
}


/**
 * @brief Convert an INT96 Spark timestamp to 64-bit timestamp
 *
 * @param[in,out] s Page state input/output
 * @param[in] src_pos Source position
 * @param[in] dst Pointer to row output data
 **/
inline __device__ void gpuOutputInt96Timestamp(volatile page_state_s *s, int src_pos, int64_t *dst)
{
    const uint8_t *src8;
    uint32_t dict_pos, dict_size = s->dict_size, ofs;
    int64_t ts;

    if (s->dict_base)
    {
        // Dictionary
        dict_pos = (s->dict_bits > 0) ? s->dict_idx[src_pos & (NZ_BFRSZ - 1)] : 0;
        src8 = s->dict_base;
    }
    else
    {
        // Plain
        dict_pos = src_pos;
        src8 = s->data_start;
    }
    dict_pos *= (uint32_t)s->dtype_len_in;
    ofs = 3 & reinterpret_cast<size_t>(src8);
    src8 -= ofs;    // align to 32-bit boundary
    ofs <<= 3;      // bytes -> bits
    if (dict_pos + 4 < dict_size)
    {
        uint3 v;
        int64_t nanos, day;
        v.x = *(const uint32_t *)(src8 + dict_pos + 0);
        v.y = *(const uint32_t *)(src8 + dict_pos + 4);
        v.z = *(const uint32_t *)(src8 + dict_pos + 8);
        if (ofs)
        {
            uint32_t next = *(const uint32_t *)(src8 + dict_pos + 12);
            v.x = __funnelshift_r(v.x, v.y, ofs);
            v.y = __funnelshift_r(v.y, v.z, ofs);
            v.z = __funnelshift_r(v.z, next, ofs);
        }
        nanos = v.y;
        nanos <<= 32;
        nanos |= v.x;
        day = v.z;
        // Convert from Julian day at noon to UTC seconds
        day = (day - 2440588) * (24 * 60 * 60); // TBD: Should be noon instead of midnight, but this matches pyarrow
        if (s->col.ts_clock_rate)
            ts = (day * s->col.ts_clock_rate) + (nanos + (499999999 / s->col.ts_clock_rate)) / (1000000000 / s->col.ts_clock_rate); // Output to desired clock rate
        else
            ts = (day * 1000000000) + nanos;
    }
    else
    {
        ts = 0;
    }
    *dst = ts;
}


/**
 * @brief Output a 64-bit timestamp
 *
 * @param[in,out] s Page state input/output
 * @param[in] src_pos Source position
 * @param[in] dst Pointer to row output data
 **/
inline __device__ void gpuOutputInt64Timestamp(volatile page_state_s *s, int src_pos, int64_t *dst)
{
    const uint8_t *src8;
    uint32_t dict_pos, dict_size = s->dict_size, ofs;
    int64_t ts;

    if (s->dict_base)
    {
        // Dictionary
        dict_pos = (s->dict_bits > 0) ? s->dict_idx[src_pos & (NZ_BFRSZ - 1)] : 0;
        src8 = s->dict_base;
    }
    else
    {
        // Plain
        dict_pos = src_pos;
        src8 = s->data_start;
    }
    dict_pos *= (uint32_t)s->dtype_len_in;
    ofs = 3 & reinterpret_cast<size_t>(src8);
    src8 -= ofs;    // align to 32-bit boundary
    ofs <<= 3;      // bytes -> bits
    if (dict_pos + 4 < dict_size)
    {
        uint2 v;
        int64_t val;
        v.x = *(const uint32_t *)(src8 + dict_pos + 0);
        v.y = *(const uint32_t *)(src8 + dict_pos + 4);
        if (ofs)
        {
            uint32_t next = *(const uint32_t *)(src8 + dict_pos + 8);
            v.x = __funnelshift_r(v.x, v.y, ofs);
            v.y = __funnelshift_r(v.y, next, ofs);
        }
        val = v.y;
        val <<= 32;
        val |= v.x;
        ts = ((val * s->int64_nanoscale) + (499999999 / s->col.ts_clock_rate)) / (1000000000 / s->col.ts_clock_rate); // Output to desired clock rate
    }
    else
    {
        ts = 0;
    }
    *dst = ts;
}


/**
 * @brief Powers of 10
 **/
static const __device__ __constant__ double kPow10[40] =
{
    1.0,    1.e1,   1.e2,   1.e3,   1.e4,   1.e5,   1.e6,   1.e7,
    1.e8,   1.e9,   1.e10,  1.e11,  1.e12,  1.e13,  1.e14,  1.e15,
    1.e16,  1.e17,  1.e18,  1.e19,  1.e20,  1.e21,  1.e22,  1.e23,
    1.e24,  1.e25,  1.e26,  1.e27,  1.e28,  1.e29,  1.e30,  1.e31,
    1.e32,  1.e33,  1.e34,  1.e35,  1.e36,  1.e37,  1.e38,  1.e39,
};

/**
 * @brief Output a decimal type ([INT32..INT128] + scale) as a 64-bit float
 *
 * @param[in,out] s Page state input/output
 * @param[in] src_pos Source position
 * @param[in] dst Pointer to row output data
 * @param[in] dtype Stored data type
 **/
inline __device__ void gpuOutputDecimal(volatile page_state_s *s, int src_pos, double *dst, int dtype)
{
    const uint8_t *dict;
    uint32_t dict_pos, dict_size = s->dict_size, dtype_len_in;
    int64_t i128_hi, i128_lo;
    int32_t scale;
    double d;

    if (s->dict_base)
    {
        // Dictionary
        dict_pos = (s->dict_bits > 0) ? s->dict_idx[src_pos & (NZ_BFRSZ - 1)] : 0;
        dict = s->dict_base;
    }
    else
    {
        // Plain
        dict_pos = src_pos;
        dict = s->data_start;
    }
    dtype_len_in = s->dtype_len_in;
    dict_pos *= dtype_len_in;
    // FIXME: Not very efficient (currently reading 1 byte at a time) -> need a variable-length unaligned
    // load utility function (both little-endian and big-endian versions)
    if (dtype == INT32)
    {
        int32_t lo32 = 0;
        for (unsigned int i = 0; i < dtype_len_in; i++) {
            uint32_t v = (dict_pos + i < dict_size) ? dict[dict_pos + i] : 0;
            lo32 |= v << (i * 8);
        }
        i128_lo = lo32;
        i128_hi = lo32 >> 31;
    }
    else if (dtype == INT64)
    {
        int64_t lo64 = 0;
        for (unsigned int i = 0; i < dtype_len_in; i++) {
            uint64_t v = (dict_pos + i < dict_size) ? dict[dict_pos + i] : 0;
            lo64 |= v << (i*8);
        }
        i128_lo = lo64;
        i128_hi = lo64 >> 63;
    }
    else // if (dtype == FIXED_LENGTH_BYTE_ARRAY)
    {
        i128_lo = 0;
        for (unsigned int i = dtype_len_in - min(dtype_len_in, 8); i < dtype_len_in; i++) {
            uint32_t v = (dict_pos + i < dict_size) ? dict[dict_pos + i] : 0;
            i128_lo = (i128_lo << 8) | v;
        }
        if (dtype_len_in > 8) {
            i128_hi = 0;
            for (unsigned int i = dtype_len_in - min(dtype_len_in, 16); i < dtype_len_in - 8; i++) {
                uint32_t v = (dict_pos + i < dict_size) ? dict[dict_pos + i] : 0;
                i128_hi = (i128_hi << 8) | v;
            }
            if (dtype_len_in < 16) {
                i128_hi <<= 64 - (dtype_len_in - 8) * 8;
                i128_hi >>= 64 - (dtype_len_in - 8) * 8;
            }
        }
        else {
            if (dtype_len_in < 8) {
                i128_lo <<= 64 - dtype_len_in * 8;
                i128_lo >>= 64 - dtype_len_in * 8;
            }
            i128_hi = i128_lo >> 63;
        }
    }
    scale = s->col.decimal_scale;
    d = Int128ToDouble_rn(i128_lo, i128_hi);
    *dst = (scale < 0) ? (d * kPow10[min(-scale, 39)]) : (d / kPow10[min(scale, 39)]);
}


/**
 * @brief Output a small fixed-length value
 *
 * @param[in,out] s Page state input/output
 * @param[in] src_pos Source position
 * @param[in] dst Pointer to row output data
 **/
template <typename T>
inline __device__ void gpuOutputFast(volatile page_state_s *s, int src_pos, T *dst)
{
    const uint8_t *dict;
    uint32_t dict_pos, dict_size = s->dict_size;

    if (s->dict_base)
    {
        // Dictionary
        dict_pos = (s->dict_bits > 0) ? s->dict_idx[src_pos & (NZ_BFRSZ - 1)] : 0;
        dict = s->dict_base;
    }
    else
    {
        // Plain
        dict_pos = src_pos;
        dict = s->data_start;
    }
    dict_pos *= (uint32_t)s->dtype_len_in;
    gpuStoreOutput(dst, dict, dict_pos, dict_size);
}


/**
 * @brief Output a N-byte value
 *
 * @param[in,out] s Page state input/output
 * @param[in] src_pos Source position
 * @param[in] dst8 Pointer to row output data
 * @param[in] len Length of element
 **/
static __device__ void gpuOutputGeneric(volatile page_state_s *s, int src_pos, uint8_t *dst8, int len)
{
    const uint8_t *dict;
    uint32_t dict_pos, dict_size = s->dict_size;
    
    if (s->dict_base)
    {
        // Dictionary
        dict_pos = (s->dict_bits > 0) ? s->dict_idx[src_pos & (NZ_BFRSZ - 1)] : 0;
        dict = s->dict_base;
    }
    else
    {
        // Plain
        dict_pos = src_pos;
        dict = s->data_start;
    }
    dict_pos *= (uint32_t)s->dtype_len_in;
    if (len & 3)
    {
        // Generic slow path
        for (unsigned int i = 0; i < len; i++)
        {
            dst8[i] = (dict_pos + i < dict_size) ? dict[dict_pos + i] : 0;
        }
    }
    else
    {
        // Copy 4 bytes at a time
        const uint8_t *src8 = dict;
        unsigned int ofs = 3 & reinterpret_cast<size_t>(src8);
        src8 -= ofs;    // align to 32-bit boundary
        ofs <<= 3;      // bytes -> bits
        for (unsigned int i = 0; i < len; i += 4)
        {
            uint32_t bytebuf;
            if (dict_pos < dict_size)
            {
                bytebuf = *(const uint32_t *)(src8 + dict_pos);
                if (ofs)
                {
                    uint32_t bytebufnext = *(const uint32_t *)(src8 + dict_pos + 4);
                    bytebuf = __funnelshift_r(bytebuf, bytebufnext, ofs);
                }
            }
            else
            {
                bytebuf = 0;
            }
            dict_pos += 4;
            *(uint32_t *)(dst8 + i) = bytebuf;
        }
    }
}


/**
 * @brief Kernel for reading the column data stored in the pages
 *
 * This function will write the page data and the page data's validity to the
 * output specified in the page's column chunk. If necessary, additional
 * conversion will be perfomed to translate from the Parquet datatype to
 * desired output datatype (ex. 32-bit to 16-bit, string to hash).
 *
 * @param[in] pages List of pages
 * @param[in,out] chunks List of column chunks
 * @param[in] min_row crop all rows below min_row
 * @param[in] num_rows Maximum number of rows to read
 * @param[in] num_chunks Number of column chunks
 **/
// blockDim {NTHREADS,1,1}
extern "C" __global__ void __launch_bounds__(NTHREADS)
gpuDecodePageData(PageInfo *pages, ColumnChunkDesc *chunks, size_t min_row, size_t num_rows, int32_t num_chunks)
{
    __shared__ __align__(16) page_state_s state_g;

    page_state_s * const s = &state_g;
    int page_idx = blockIdx.x;
    int t = threadIdx.x;
    int chunk_idx, out_thread0;
    
    // Fetch page info
    // NOTE: Assumes that sizeof(PageInfo) <= 256
    if (t < sizeof(PageInfo) / sizeof(uint32_t))
    {
        ((uint32_t *)&s->page)[t] = ((const uint32_t *)&pages[page_idx])[t];
    }
    __syncthreads();
    if (s->page.flags & PAGEINFO_FLAGS_DICTIONARY)
    {
        return;
    }
    // Fetch column chunk info
    chunk_idx = s->page.chunk_idx;
    if ((uint32_t)chunk_idx < (uint32_t)num_chunks)
    {
        // NOTE: Assumes that sizeof(ColumnChunkDesc) <= 256
        if (t < sizeof(ColumnChunkDesc) / sizeof(uint32_t))
        {
            ((uint32_t *)&s->col)[t] = ((const uint32_t *)&chunks[chunk_idx])[t];
        }
    }
    __syncthreads();
    if (!t)
    {
        s->num_rows = 0;
        s->page.valid_count = 0;
        s->error = 0;
        if (s->page.num_values > 0 && s->page.num_rows > 0)
        {
            uint8_t *cur = s->page.page_data;
            uint8_t *end = cur + s->page.uncompressed_page_size;
            size_t page_start_row = s->col.start_row + s->page.chunk_row;
            uint32_t dtype_len_out = s->col.data_type >> 3;
            s->int64_nanoscale = 0;
            // Validate data type
            switch(s->col.data_type & 7)
            {
            case BOOLEAN:
                s->dtype_len = 1;  // Boolean are stored as 1 byte on the output
                break;
            case INT32:
            case FLOAT:
                s->dtype_len = 4;
                break;
            case INT64:
                if (s->col.ts_clock_rate)
                {
                    if (s->col.converted_type == TIME_MICROS || s->col.converted_type == TIMESTAMP_MICROS)
                        if (s->col.ts_clock_rate != 1000000)
                            s->int64_nanoscale = 1000;
                    else if (s->col.converted_type == TIME_MILLIS || s->col.converted_type == TIMESTAMP_MILLIS)
                        if (s->col.ts_clock_rate != 1000)
                            s->int64_nanoscale = 1000000;
                }
                // Fall through to DOUBLE
            case DOUBLE:
                s->dtype_len = 8;
                break;
            case INT96:
                s->dtype_len = 12;
                break;
            case BYTE_ARRAY:
                s->dtype_len = sizeof(nvstrdesc_s);
                break;
            default: // FIXED_LEN_BYTE_ARRAY:
                s->dtype_len = dtype_len_out;
                s->error |= (s->dtype_len <= 0);
                break;
            }
            // Special check for downconversions
            s->dtype_len_in = s->dtype_len;
            if (s->col.converted_type == DECIMAL)
            {
                s->dtype_len = 8; // Convert DECIMAL to 64-bit float
            }
            else if ((s->col.data_type & 7) == INT32)
            {
                if (dtype_len_out == 1)
                    s->dtype_len = 1; // INT8 output
                if (dtype_len_out == 2)
                    s->dtype_len = 2; // INT16 output
            }
            else if ((s->col.data_type & 7) == BYTE_ARRAY && dtype_len_out == 4)
            {
                s->dtype_len = 4; // HASH32 output
            }
            else if ((s->col.data_type & 7) == INT96)
            {
                s->dtype_len = 8; // Convert to 64-bit timestamp
            }
            // Setup local valid map and compute first & num rows relative to the current page
            s->data_out = reinterpret_cast<uint8_t *>(s->col.column_data_base);
            s->valid_map = s->col.valid_map_base;
            s->valid_map_offset = 0;
            if (page_start_row >= min_row)
            {
                if (s->data_out)
                {
                    s->data_out += (page_start_row - min_row) * s->dtype_len;
                }
                if (s->valid_map)
                {
                    s->valid_map += (page_start_row - min_row) >> 5;
                    s->valid_map_offset = (int32_t)((page_start_row - min_row) & 0x1f);
                }
                s->first_row = 0;
            }
            else // First row starts after the beginning of the page
            {
                s->first_row = (int32_t)min(min_row - page_start_row, (size_t)s->page.num_rows);
            }
            s->num_rows = s->page.num_rows;
            s->out_valid = 0;
            s->out_valid_mask = (~0) << s->valid_map_offset;
            if (page_start_row + s->num_rows > min_row + num_rows)
            {
                s->num_rows = (int32_t)max((int64_t)(min_row + num_rows - page_start_row), INT64_C(0));
            }
            // Find the compressed size of definition levels
            cur += InitLevelSection(s, cur, end, s->page.definition_level_encoding, s->col.def_level_bits, 0);
            // Find the compressed size of repetition levels
            cur += InitLevelSection(s, cur, end, s->page.repetition_level_encoding, s->col.rep_level_bits, 1);
            s->dict_bits = 0;
            s->dict_base = 0;
            s->dict_size = 0;
            switch (s->page.encoding)
            {
            case PLAIN_DICTIONARY:
            case RLE_DICTIONARY:
                // RLE-packed dictionary indices, first byte indicates index length in bits
                if (((s->col.data_type & 7) == BYTE_ARRAY) && (s->col.str_dict_index))
                {
                    // String dictionary: use index
                    s->dict_base = reinterpret_cast<const uint8_t *>(s->col.str_dict_index);
                    s->dict_size = s->col.page_info[0].num_values * sizeof(nvstrdesc_s);
                }
                else
                {
                    s->dict_base = s->col.page_info[0].page_data; // dictionary is always stored in the first page
                    s->dict_size = s->col.page_info[0].uncompressed_page_size;
                }
                s->dict_run = 0;
                s->dict_val = 0;
                s->dict_bits = (cur < end) ? *cur++ : 0;
                if (s->dict_bits > 32 || !s->dict_base)
                {
                    s->error = (10 << 8) | s->dict_bits;
                }
                break;
            case PLAIN:
                s->dict_size = static_cast<int32_t>(end - cur);
                s->dict_val = 0;
                if ((s->col.data_type & 7) == BOOLEAN)
                {
                    s->dict_run = s->dict_size * 2 + 1;
                }
                break;
            case RLE:
                s->dict_run = 0;
                break;
            default:
                s->error = 1;   // Unsupported encoding
                break;
            }
            if (cur > end)
            {
                s->error = 1;
            }
            s->data_start = cur;
            s->data_end = end;
        }
        else
        {
            s->error = 1;
        }
        s->value_count = 0;
        s->nz_count = 0;
        s->dict_pos = 0;
        s->out_pos = 0;
        s->num_values = min(s->page.num_values, s->num_rows);
        __threadfence_block();
    }
    __syncthreads();
    if (s->dict_base)
    {
        out_thread0 = (s->dict_bits > 0) ? 64 : 32;
    }
    else
    {
        out_thread0 = ((s->col.data_type & 7) == BOOLEAN || (s->col.data_type & 7) == BYTE_ARRAY) ? 64 : 32;
    }

    while (!s->error && (s->value_count < s->num_values || s->out_pos < s->nz_count))
    {
        int target_pos;

        if (t < out_thread0)
        {
            target_pos = min(s->out_pos + 2 * (NTHREADS - out_thread0), s->nz_count + (NTHREADS - out_thread0));
        }
        else
        {
            target_pos = min(s->nz_count, s->out_pos + NTHREADS - out_thread0);
            if (out_thread0 > 32)
            {
                target_pos = min(target_pos, s->dict_pos);
            }
        }
        __syncthreads();
        if (t < 32)
        {
            // WARP0: Decode definition and repetition levels, outputs row indices
            gpuDecodeLevels(s, target_pos, t);
        }
        else if (t < out_thread0)
        {
            // WARP1: Decode dictionary indices, booleans or string positions
            if (s->dict_base)
            {
                target_pos = gpuDecodeDictionaryIndices(s, target_pos, t & 0x1f);
            }
            else if ((s->col.data_type & 7) == BOOLEAN)
            {
                target_pos = gpuDecodeRleBooleans(s, target_pos, t & 0x1f);
            }
            else if ((s->col.data_type & 7) == BYTE_ARRAY)
            {
                gpuInitStringDescriptors(s, target_pos, t & 0x1f);
            }
            if (t == 32)
            {
                *(volatile int32_t *)&s->dict_pos = target_pos;
            }
        }
        else
        {
            // WARP1..WARP3: Decode values
            int dtype = s->col.data_type & 7;
            int out_pos = s->out_pos + t - out_thread0;
            int row_idx = s->nz_idx[out_pos & (NZ_BFRSZ - 1)];
            if (out_pos < target_pos && row_idx >= 0 && s->first_row + row_idx < s->num_rows)
            {
                uint32_t dtype_len = s->dtype_len;
                uint8_t *dst = s->data_out + (size_t)row_idx * dtype_len;
                if (dtype == BYTE_ARRAY)
                    gpuOutputString(s, out_pos, dst);
                else if (dtype == BOOLEAN)
                    gpuOutputBoolean(s, out_pos, dst);
                else if (s->col.converted_type == DECIMAL)
                    gpuOutputDecimal(s, out_pos, reinterpret_cast<double *>(dst), dtype);
                else if (dtype == INT96)
                    gpuOutputInt96Timestamp(s, out_pos, reinterpret_cast<int64_t *>(dst));
                else if (dtype_len == 8)
                {
                    if (s->int64_nanoscale)
                        gpuOutputInt64Timestamp(s, out_pos, reinterpret_cast<int64_t *>(dst));
                    else
                        gpuOutputFast(s, out_pos, reinterpret_cast<uint2 *>(dst));
                }
                else if (dtype_len == 4)
                    gpuOutputFast(s, out_pos, reinterpret_cast<uint32_t *>(dst));
                else
                    gpuOutputGeneric(s, out_pos, dst, dtype_len);
            }
            if (t == out_thread0)
            {
                *(volatile int32_t *)&s->out_pos = target_pos;
            }
        }
        __syncthreads();
    }
    __syncthreads();
    if (!t)
    {
        // Update the number of rows (after cropping to [min_row, min_row+num_rows-1]), and number of valid values
        pages[page_idx].num_rows = s->num_rows - s->first_row;
        pages[page_idx].valid_count = (s->error) ? -s->error : s->page.valid_count;
    }
}


hipError_t __host__ DecodePageData(PageInfo *pages, int32_t num_pages,
                                    ColumnChunkDesc *chunks, int32_t num_chunks,
                                    size_t num_rows, size_t min_row,
                                    hipStream_t stream) {
  dim3 dim_block(NTHREADS, 1);
  dim3 dim_grid(num_pages, 1);  // 1 threadblock per page
  gpuDecodePageData <<< dim_grid, dim_block, 0, stream >>> (pages, chunks, min_row, num_rows, num_chunks);
  return hipSuccess;
}

} // namespace gpu
} // namespace parquet
} // namespace io
} // namespace cudf
