#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2018-2020, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <thrust/tuple.h>
#include <io/parquet/parquet_gpu.hpp>
#include <io/utilities/block_utils.cuh>

#include <rmm/cuda_stream_view.hpp>

namespace cudf {
namespace io {
namespace parquet {
namespace gpu {
// Minimal thrift implementation for parsing page headers
// https://github.com/apache/thrift/blob/master/doc/specs/thrift-compact-protocol.md

static const __device__ __constant__ uint8_t g_list2struct[16] = {0,
                                                                  1,
                                                                  2,
                                                                  ST_FLD_BYTE,
                                                                  ST_FLD_DOUBLE,
                                                                  5,
                                                                  ST_FLD_I16,
                                                                  7,
                                                                  ST_FLD_I32,
                                                                  9,
                                                                  ST_FLD_I64,
                                                                  ST_FLD_BINARY,
                                                                  ST_FLD_STRUCT,
                                                                  ST_FLD_MAP,
                                                                  ST_FLD_SET,
                                                                  ST_FLD_LIST};

struct byte_stream_s {
  const uint8_t *cur;
  const uint8_t *end;
  const uint8_t *base;
  // Parsed symbols
  PageType page_type;
  PageInfo page;
  ColumnChunkDesc ck;
};

/**
 * @brief Get current byte from the byte stream
 *
 * @param[in] bs Byte stream
 *
 * @return Current byte pointed to by the byte stream
 */
inline __device__ unsigned int getb(byte_stream_s *bs)
{
  return (bs->cur < bs->end) ? *bs->cur++ : 0;
}

inline __device__ void skip_bytes(byte_stream_s *bs, size_t bytecnt)
{
  bytecnt = min(bytecnt, (size_t)(bs->end - bs->cur));
  bs->cur += bytecnt;
}

/**
 * @brief Decode unsigned integer from a byte stream using VarInt encoding
 *
 * Concatenate least significant 7 bits of each byte to form a 32 bit
 * integer. Most significant bit of each byte indicates if more bytes
 * are to be used to form the number.
 *
 * @param[in] bs Byte stream
 *
 * @return Decoded 32 bit integer
 */
__device__ uint32_t get_u32(byte_stream_s *bs)
{
  uint32_t v = 0, l = 0, c;
  do {
    c = getb(bs);
    v |= (c & 0x7f) << l;
    l += 7;
  } while (c & 0x80);
  return v;
}

/**
 * @brief Decode signed integer from a byte stream using zigzag encoding
 *
 * The number n encountered in a byte stream translates to
 * -1^(n%2) * ceil(n/2), with the exception of 0 which remains the same.
 * i.e. 0, 1, 2, 3, 4, 5 etc convert to 0, -1, 1, -2, 2 respectively.
 *
 * @param[in] bs Byte stream
 *
 * @return Decoded 32 bit integer
 */
inline __device__ int32_t get_i32(byte_stream_s *bs)
{
  uint32_t u = get_u32(bs);
  return (int32_t)((u >> 1u) ^ -(int32_t)(u & 1));
}

__device__ void skip_struct_field(byte_stream_s *bs, int field_type)
{
  int struct_depth = 0;
  int rep_cnt      = 0;

  do {
    if (rep_cnt != 0) {
      rep_cnt--;
    } else if (struct_depth != 0) {
      unsigned int c;
      do {
        c = getb(bs);
        if (!c) --struct_depth;
      } while (!c && struct_depth);
      if (!struct_depth) break;
      field_type = c & 0xf;
      if (!(c & 0xf0)) get_i32(bs);
    }
    switch (field_type) {
      case ST_FLD_TRUE:
      case ST_FLD_FALSE: break;
      case ST_FLD_I16:
      case ST_FLD_I32:
      case ST_FLD_I64: get_u32(bs); break;
      case ST_FLD_BYTE: skip_bytes(bs, 1); break;
      case ST_FLD_DOUBLE: skip_bytes(bs, 8); break;
      case ST_FLD_BINARY: skip_bytes(bs, get_u32(bs)); break;
      case ST_FLD_LIST:
      case ST_FLD_SET: {  // NOTE: skipping a list of lists is not handled
        auto const c = getb(bs);
        int n        = c >> 4;
        if (n == 0xf) n = get_u32(bs);
        field_type = g_list2struct[c & 0xf];
        if (field_type == ST_FLD_STRUCT)
          struct_depth += n;
        else
          rep_cnt = n;
      } break;
      case ST_FLD_STRUCT: struct_depth++; break;
    }
  } while (rep_cnt || struct_depth);
}

/**
 * @brief Functor to set value to 32 bit integer read from byte stream
 *
 * @return True if field type is not int32
 */
struct ParquetFieldInt32 {
  int field;
  int32_t &val;

  __device__ ParquetFieldInt32(int f, int32_t &v) : field(f), val(v) {}

  inline __device__ bool operator()(byte_stream_s *bs, int field_type)
  {
    val = get_i32(bs);
    return (field_type != ST_FLD_I32);
  }
};

/**
 * @brief Functor to set value to enum read from byte stream
 *
 * @return True if field type is not int32
 */
template <typename Enum>
struct ParquetFieldEnum {
  int field;
  Enum &val;

  __device__ ParquetFieldEnum(int f, Enum &v) : field(f), val(v) {}

  inline __device__ bool operator()(byte_stream_s *bs, int field_type)
  {
    val = static_cast<Enum>(get_i32(bs));
    return (field_type != ST_FLD_I32);
  }
};

/**
 * @brief Functor to run operator on byte stream
 *
 * @return True if field type is not struct type or if the calling operator
 * fails
 */
template <typename Operator>
struct ParquetFieldStruct {
  int field;
  Operator op;

  __device__ ParquetFieldStruct(int f) : field(f) {}

  inline __device__ bool operator()(byte_stream_s *bs, int field_type)
  {
    return ((field_type != ST_FLD_STRUCT) || !op(bs));
  }
};

/**
 * @brief Functor to run an operator
 *
 * The purpose of this functor is to replace a switch case. If the field in
 * the argument is equal to the field specified in any element of the tuple
 * of operators then it is run with the byte stream and field type arguments.
 *
 * If the field does not match any of the functors then skip_struct_field is
 * called over the byte stream.
 *
 * @return Return value of the selected operator or false if no operator
 * matched the field value
 */
template <int index>
struct FunctionSwitchImpl {
  template <typename... Operator>
  static inline __device__ bool run(byte_stream_s *bs,
                                    int field_type,
                                    const int &field,
                                    thrust::tuple<Operator...> &ops)
  {
    if (field == thrust::get<index>(ops).field) {
      return thrust::get<index>(ops)(bs, field_type);
    } else {
      return FunctionSwitchImpl<index - 1>::run(bs, field_type, field, ops);
    }
  }
};

template <>
struct FunctionSwitchImpl<0> {
  template <typename... Operator>
  static inline __device__ bool run(byte_stream_s *bs,
                                    int field_type,
                                    const int &field,
                                    thrust::tuple<Operator...> &ops)
  {
    if (field == thrust::get<0>(ops).field) {
      return thrust::get<0>(ops)(bs, field_type);
    } else {
      skip_struct_field(bs, field_type);
      return false;
    }
  }
};

/**
 * @brief Function to parse page header based on the tuple of functors provided
 *
 * Bytes are read from the byte stream and the field delta and field type are
 * matched up against user supplied reading functors. If they match then the
 * corresponding values are written to references pointed to by the functors.
 *
 * @return Returns false if an unexpected field is encountered while reading
 * byte stream. Otherwise true is returned.
 */
template <typename... Operator>
inline __device__ bool parse_header(thrust::tuple<Operator...> &op, byte_stream_s *bs)
{
  constexpr int index = thrust::tuple_size<thrust::tuple<Operator...>>::value - 1;
  int field           = 0;
  while (true) {
    auto const current_byte = getb(bs);
    if (!current_byte) break;
    int const field_delta = current_byte >> 4;
    int const field_type  = current_byte & 0xf;
    field                 = field_delta ? field + field_delta : get_i32(bs);
    bool exit_function    = FunctionSwitchImpl<index>::run(bs, field_type, field, op);
    if (exit_function) { return false; }
  }
  return true;
}

struct gpuParseDataPageHeader {
  __device__ bool operator()(byte_stream_s *bs)
  {
    auto op = thrust::make_tuple(ParquetFieldInt32(1, bs->page.num_input_values),
                                 ParquetFieldEnum<Encoding>(2, bs->page.encoding),
                                 ParquetFieldEnum<Encoding>(3, bs->page.definition_level_encoding),
                                 ParquetFieldEnum<Encoding>(4, bs->page.repetition_level_encoding));
    return parse_header(op, bs);
  }
};

struct gpuParseDictionaryPageHeader {
  __device__ bool operator()(byte_stream_s *bs)
  {
    auto op = thrust::make_tuple(ParquetFieldInt32(1, bs->page.num_input_values),
                                 ParquetFieldEnum<Encoding>(2, bs->page.encoding));
    return parse_header(op, bs);
  }
};

struct gpuParseDataPageHeaderV2 {
  __device__ bool operator()(byte_stream_s *bs)
  {
    auto op = thrust::make_tuple(ParquetFieldInt32(1, bs->page.num_input_values),
                                 ParquetFieldInt32(3, bs->page.num_rows),
                                 ParquetFieldEnum<Encoding>(4, bs->page.encoding),
                                 ParquetFieldEnum<Encoding>(5, bs->page.definition_level_encoding),
                                 ParquetFieldEnum<Encoding>(6, bs->page.repetition_level_encoding));
    return parse_header(op, bs);
  }
};

struct gpuParsePageHeader {
  __device__ bool operator()(byte_stream_s *bs)
  {
    auto op = thrust::make_tuple(ParquetFieldEnum<PageType>(1, bs->page_type),
                                 ParquetFieldInt32(2, bs->page.uncompressed_page_size),
                                 ParquetFieldInt32(3, bs->page.compressed_page_size),
                                 ParquetFieldStruct<gpuParseDataPageHeader>(5),
                                 ParquetFieldStruct<gpuParseDictionaryPageHeader>(7),
                                 ParquetFieldStruct<gpuParseDataPageHeaderV2>(8));
    return parse_header(op, bs);
  }
};

/**
 * @brief Kernel for outputting page headers from the specified column chunks
 *
 * @param[in] chunks List of column chunks
 * @param[in] num_chunks Number of column chunks
 */
// blockDim {128,1,1}
extern "C" __global__ void __launch_bounds__(128)
  gpuDecodePageHeaders(ColumnChunkDesc *chunks, int32_t num_chunks)
{
  gpuParsePageHeader parse_page_header;
  __shared__ byte_stream_s bs_g[4];

  int lane_id             = threadIdx.x % 32;
  int chunk               = (blockIdx.x * 4) + (threadIdx.x / 32);
  byte_stream_s *const bs = &bs_g[threadIdx.x / 32];

  if (chunk < num_chunks and lane_id == 0) bs->ck = chunks[chunk];
  __syncthreads();

  if (chunk < num_chunks) {
    size_t num_values, values_found;
    uint32_t data_page_count       = 0;
    uint32_t dictionary_page_count = 0;
    int32_t max_num_pages;
    int32_t num_dict_pages = bs->ck.num_dict_pages;
    PageInfo *page_info;

    if (!lane_id) {
      bs->base = bs->cur      = bs->ck.compressed_data;
      bs->end                 = bs->base + bs->ck.compressed_size;
      bs->page.chunk_idx      = chunk;
      bs->page.src_col_schema = bs->ck.src_col_schema;
      // this computation is only valid for flat schemas. for nested schemas,
      // they will be recomputed in the preprocess step by examining repetition and
      // definition levels
      bs->page.chunk_row = 0;
      bs->page.num_rows  = 0;
    }
    num_values     = bs->ck.num_values;
    page_info      = bs->ck.page_info;
    num_dict_pages = bs->ck.num_dict_pages;
    max_num_pages  = (page_info) ? bs->ck.max_num_pages : 0;
    values_found   = 0;
    __syncwarp();
    while (values_found < num_values && bs->cur < bs->end) {
      int index_out = -1;

      if (lane_id == 0) {
        // this computation is only valid for flat schemas. for nested schemas,
        // they will be recomputed in the preprocess step by examining repetition and
        // definition levels
        bs->page.chunk_row += bs->page.num_rows;
        bs->page.num_rows = 0;
        if (parse_page_header(bs) && bs->page.compressed_page_size >= 0) {
          switch (bs->page_type) {
            case PageType::DATA_PAGE:
              // this computation is only valid for flat schemas. for nested schemas,
              // they will be recomputed in the preprocess step by examining repetition and
              // definition levels
              bs->page.num_rows = bs->page.num_input_values;
            case PageType::DATA_PAGE_V2:
              index_out = num_dict_pages + data_page_count;
              data_page_count++;
              bs->page.flags = 0;
              values_found += bs->page.num_input_values;
              break;
            case PageType::DICTIONARY_PAGE:
              index_out = dictionary_page_count;
              dictionary_page_count++;
              bs->page.flags = PAGEINFO_FLAGS_DICTIONARY;
              break;
            default: index_out = -1; break;
          }
          bs->page.page_data = const_cast<uint8_t *>(bs->cur);
          bs->cur += bs->page.compressed_page_size;
        } else {
          bs->cur = bs->end;
        }
      }
      index_out = shuffle(index_out);
      if (index_out >= 0 && index_out < max_num_pages && lane_id == 0)
        page_info[index_out] = bs->page;
      num_values = shuffle(num_values);
      __syncwarp();
    }
    if (lane_id == 0) {
      chunks[chunk].num_data_pages = data_page_count;
      chunks[chunk].num_dict_pages = dictionary_page_count;
    }
  }
}

/**
 * @brief Kernel for building dictionary index for the specified column chunks
 *
 * This function builds an index to point to each dictionary entry
 * (string format is 4-byte little-endian string length followed by character
 * data). The index is a 32-bit integer which contains the offset of each string
 * relative to the beginning of the dictionary page data.
 *
 * @param[in] chunks List of column chunks
 * @param[in] num_chunks Number of column chunks
 */
// blockDim {128,1,1}
extern "C" __global__ void __launch_bounds__(128)
  gpuBuildStringDictionaryIndex(ColumnChunkDesc *chunks, int32_t num_chunks)
{
  __shared__ ColumnChunkDesc chunk_g[4];

  int lane_id               = threadIdx.x % 32;
  int chunk                 = (blockIdx.x * 4) + (threadIdx.x / 32);
  ColumnChunkDesc *const ck = &chunk_g[threadIdx.x / 32];
  if (chunk < num_chunks and lane_id == 0) *ck = chunks[chunk];
  __syncthreads();

  if (chunk >= num_chunks) { return; }
  if (!lane_id && ck->num_dict_pages > 0 && ck->str_dict_index) {
    // Data type to describe a string
    nvstrdesc_s *dict_index = ck->str_dict_index;
    const uint8_t *dict     = ck->page_info[0].page_data;
    int dict_size           = ck->page_info[0].uncompressed_page_size;
    int num_entries         = ck->page_info[0].num_input_values;
    int pos = 0, cur = 0;
    for (int i = 0; i < num_entries; i++) {
      int len = 0;
      if (cur + 4 <= dict_size) {
        len = dict[cur + 0] | (dict[cur + 1] << 8) | (dict[cur + 2] << 16) | (dict[cur + 3] << 24);
        if (len >= 0 && cur + 4 + len <= dict_size) {
          pos = cur;
          cur = cur + 4 + len;
        } else {
          cur = dict_size;
        }
      }
      // TODO: Could store 8 entries in shared mem, then do a single warp-wide store
      dict_index[i].ptr   = reinterpret_cast<const char *>(dict + pos + 4);
      dict_index[i].count = len;
    }
  }
}

void __host__ DecodePageHeaders(ColumnChunkDesc *chunks,
                                int32_t num_chunks,
                                rmm::cuda_stream_view stream)
{
  dim3 dim_block(128, 1);
  dim3 dim_grid((num_chunks + 3) >> 2, 1);  // 1 chunk per warp, 4 warps per block
  gpuDecodePageHeaders<<<dim_grid, dim_block, 0, stream.value()>>>(chunks, num_chunks);
}

void __host__ BuildStringDictionaryIndex(ColumnChunkDesc *chunks,
                                         int32_t num_chunks,
                                         rmm::cuda_stream_view stream)
{
  dim3 dim_block(128, 1);
  dim3 dim_grid((num_chunks + 3) >> 2, 1);  // 1 chunk per warp, 4 warps per block
  gpuBuildStringDictionaryIndex<<<dim_grid, dim_block, 0, stream.value()>>>(chunks, num_chunks);
}

}  // namespace gpu
}  // namespace parquet
}  // namespace io
}  // namespace cudf
