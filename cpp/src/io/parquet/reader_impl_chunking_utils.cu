#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2025, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "io/comp/decompression.hpp"
#include "io/comp/gpuinflate.hpp"
#include "io/utilities/time_utils.cuh"
#include "reader_impl_chunking.hpp"
#include "reader_impl_chunking_utils.cuh"

#include <cudf/detail/iterator.cuh>
#include <cudf/detail/nvtx/ranges.hpp>
#include <cudf/detail/utilities/vector_factories.hpp>
#include <cudf/io/parquet.hpp>
#include <cudf/utilities/memory_resource.hpp>

#include <rmm/exec_policy.hpp>

#include <cub/device/device_radix_sort.cuh>
#include <thrust/binary_search.h>
#include <thrust/iterator/constant_iterator.h>
#include <thrust/iterator/discard_iterator.h>
#include <thrust/logical.h>
#include <thrust/sequence.h>
#include <thrust/transform_scan.h>
#include <thrust/unique.h>

#include <algorithm>
#include <array>
#include <iostream>
#include <numeric>

namespace cudf::io::parquet::detail {

using cudf::io::detail::codec_exec_result;
using cudf::io::detail::codec_status;
using cudf::io::detail::decompression_info;

#if defined(CHUNKING_DEBUG)
void print_cumulative_page_info(device_span<PageInfo const> d_pages,
                                device_span<ColumnChunkDesc const> d_chunks,
                                device_span<cumulative_page_info const> d_c_info,
                                rmm::cuda_stream_view stream)
{
  auto const pages  = cudf::detail::make_host_vector(d_pages, stream);
  auto const chunks = cudf::detail::make_host_vector(d_chunks, stream);
  auto const c_info = cudf::detail::make_host_vector(d_c_info, stream);

  std::cout << "------------\nCumulative sizes by page\n";

  std::vector<int> schemas(pages.size());
  auto schema_iter = cudf::detail::make_counting_transform_iterator(
    0, [&](size_type i) { return pages[i].src_col_schema; });
  thrust::copy(thrust::seq, schema_iter, schema_iter + pages.size(), schemas.begin());
  auto last = thrust::unique(thrust::seq, schemas.begin(), schemas.end());
  schemas.resize(last - schemas.begin());
  std::cout << "Num schemas: " << schemas.size() << "\n";

  for (size_t idx = 0; idx < schemas.size(); idx++) {
    std::cout << "Schema " << schemas[idx] << ":\n";
    for (size_t pidx = 0; pidx < pages.size(); pidx++) {
      auto const& page = pages[pidx];
      if (page.flags & PAGEINFO_FLAGS_DICTIONARY || page.src_col_schema != schemas[idx]) {
        continue;
      }
      bool const is_list = chunks[page.chunk_idx].max_level[level_type::REPETITION] > 0;
      std::cout << "\tP " << (is_list ? "(L) " : "") << "{" << pidx << ", "
                << c_info[pidx].end_row_index << ", " << c_info[pidx].size_bytes << "}\n";
    }
  }
}

void print_cumulative_page_info(host_span<cumulative_page_info const> sizes,
                                std::string const& label,
                                std::optional<std::vector<row_range>> splits = std::nullopt)
{
  if (splits.has_value()) {
    std::cout << "------------\nSplits (skip_rows, num_rows)\n";
    for (size_t idx = 0; idx < splits->size(); idx++) {
      std::cout << "{" << splits.value()[idx].skip_rows << ", " << splits.value()[idx].num_rows
                << "}\n";
    }
  }

  std::cout << "------------\nCumulative sizes " << label.c_str()
            << " (index, row_index, size_bytes, page_key)\n";
  for (size_t idx = 0; idx < sizes.size(); idx++) {
    std::cout << "{" << idx << ", " << sizes[idx].end_row_index << ", " << sizes[idx].size_bytes
              << ", " << sizes[idx].key << "}";

    if (splits.has_value()) {
      // if we have a split at this row count and this is the last instance of this row count
      auto start             = thrust::make_transform_iterator(splits->begin(),
                                                   [](row_range const& i) { return i.skip_rows; });
      auto end               = start + splits->size();
      auto split             = std::find(start, end, sizes[idx].end_row_index);
      auto const split_index = [&]() -> int {
        if (split != end && ((idx == sizes.size() - 1) ||
                             (sizes[idx + 1].end_row_index > sizes[idx].end_row_index))) {
          return static_cast<int>(std::distance(start, split));
        }
        return idx == 0 ? 0 : -1;
      }();
      if (split_index >= 0) {
        std::cout << " <-- split {" << splits.value()[split_index].skip_rows << ", "
                  << splits.value()[split_index].num_rows << "}";
      }
    }
    std::cout << "\n";
  }
}
#endif  // CHUNKING_DEBUG

void codec_stats::add_pages(host_span<ColumnChunkDesc const> chunks,
                            host_span<PageInfo> pages,
                            page_selection selection,
                            host_span<bool const> page_mask)
{
  // Create a page mask iterator that defaults to true if the page_mask is empty
  auto page_mask_iter = thrust::make_constant_iterator(true);

  // Zip iterator for iterating over pages and the page mask
  auto zip_iter = thrust::make_zip_iterator(pages.begin(), page_mask_iter);

  std::for_each(zip_iter, zip_iter + pages.size(), [&](auto const& item) {
    auto& [page, is_page_needed] = item;
    // If this is a V2 page, use the `is_compressed` field to determine if it's compressed.
    // For V1 pages, it's always compressed if the chunk.codec is specified.
    auto const is_page_compressed = (page.flags & PAGEINFO_FLAGS_V2) ? page.is_compressed : true;
    if (is_page_needed && chunks[page.chunk_idx].codec == compression_type &&
        (page.flags & cudf::io::parquet::detail::PAGEINFO_FLAGS_DICTIONARY) ==
          (selection == page_selection::DICT_PAGES) and
        is_page_compressed) {
      ++num_pages;
      total_decomp_size += page.uncompressed_page_size;
      max_decompressed_size = std::max(max_decompressed_size, page.uncompressed_page_size);
    }
  });
}

CUDF_HOST_DEVICE cuda::std::pair<compression_type, bool> parquet_compression_support(
  Compression compression)
{
  switch (compression) {
    case Compression::BROTLI: return {compression_type::BROTLI, true};
    case Compression::GZIP: return {compression_type::GZIP, true};
    case Compression::LZ4_RAW: return {compression_type::LZ4, true};
    case Compression::LZO: return {compression_type::LZO, false};
    case Compression::SNAPPY: return {compression_type::SNAPPY, true};
    case Compression::ZSTD: return {compression_type::ZSTD, true};
    case Compression::UNCOMPRESSED: return {compression_type::NONE, true};
    default: break;
  }
  return {compression_type::NONE, false};
}

[[nodiscard]] std::string parquet_compression_name(Compression compression)
{
  switch (compression) {
    case Compression::BROTLI: return "BROTLI";
    case Compression::GZIP: return "GZIP";
    case Compression::LZ4_RAW: return "LZ4_RAW";
    case Compression::LZ4: return "LZ4";
    case Compression::LZO: return "LZO";
    case Compression::SNAPPY: return "SNAPPY";
    case Compression::ZSTD: return "ZSTD";
    case Compression::UNCOMPRESSED: return "UNCOMPRESSED";
  }
  CUDF_FAIL("Unsupported Parquet compression type");
}

compression_type from_parquet_compression(Compression compression)
{
  auto const [type, supported] = parquet_compression_support(compression);
  CUDF_EXPECTS(supported,
               "Unsupported Parquet compression type: " + parquet_compression_name(compression));
  return type;
}

size_t find_start_index(cudf::host_span<cumulative_page_info const> aggregated_info,
                        size_t start_row)
{
  auto start = thrust::make_transform_iterator(
    aggregated_info.begin(), [&](cumulative_page_info const& i) { return i.end_row_index; });
  return thrust::lower_bound(thrust::host, start, start + aggregated_info.size(), start_row) -
         start;
}

int64_t find_next_split(int64_t cur_pos,
                        size_t cur_row_index,
                        size_t cur_cumulative_size,
                        cudf::host_span<cumulative_page_info const> sizes,
                        size_t size_limit,
                        size_t min_row_count)
{
  auto const start = thrust::make_transform_iterator(
    sizes.begin(),
    [&](cumulative_page_info const& i) { return i.size_bytes - cur_cumulative_size; });
  auto const end = start + sizes.size();

  int64_t split_pos = thrust::lower_bound(thrust::seq, start + cur_pos, end, size_limit) - start;

  // if we're past the end, or if the returned bucket is > than the chunk_read_limit, move back
  // one as long as this doesn't put us before our starting point.
  if (static_cast<size_t>(split_pos) >= sizes.size() ||
      ((split_pos > cur_pos) && (sizes[split_pos].size_bytes - cur_cumulative_size > size_limit))) {
    split_pos--;
  }

  // move forward until we find the next group of pages that will actually advance our row count.
  // this guarantees that even if we cannot fit the set of rows represented by our where our cur_pos
  // is, we will still move forward instead of failing.
  while (split_pos < (static_cast<int64_t>(sizes.size()) - 1) &&
         (sizes[split_pos].end_row_index - cur_row_index < min_row_count)) {
    split_pos++;
  }

  return split_pos;
}

[[nodiscard]] std::tuple<int32_t, std::optional<LogicalType>> conversion_info(
  type_id column_type_id,
  type_id timestamp_type_id,
  Type physical,
  std::optional<LogicalType> logical_type)
{
  int32_t const clock_rate =
    is_chrono(data_type{column_type_id}) ? to_clockrate(timestamp_type_id) : 0;

  // TODO(ets): this is leftover from the original code, but will we ever output decimal as
  // anything but fixed point?
  if (logical_type.has_value() and logical_type->type == LogicalType::DECIMAL) {
    // if decimal but not outputting as float or decimal, then convert to no logical type
    if (column_type_id != type_id::FLOAT64 and
        not cudf::is_fixed_point(data_type{column_type_id})) {
      return {clock_rate, std::nullopt};
    }
  }

  return {clock_rate, std::move(logical_type)};
}

std::pair<size_t, size_t> get_row_group_size(RowGroup const& rg)
{
  auto compressed_size_iter = thrust::make_transform_iterator(
    rg.columns.begin(), [](ColumnChunk const& c) { return c.meta_data.total_compressed_size; });

  // the trick is that total temp space needed is tricky to know
  auto const compressed_size =
    std::reduce(compressed_size_iter, compressed_size_iter + rg.columns.size());
  auto const total_size = compressed_size + rg.total_byte_size;
  return {compressed_size, total_size};
}

std::pair<rmm::device_uvector<cumulative_page_info>, rmm::device_uvector<int32_t>>
adjust_cumulative_sizes(device_span<cumulative_page_info const> c_info,
                        device_span<PageInfo const> pages,
                        rmm::cuda_stream_view stream)
{
  // sort by row count
  rmm::device_uvector<cumulative_page_info> c_info_sorted(c_info.size(), stream);
  {
    rmm::device_uvector<size_t> end_row_indices(c_info.size(), stream);
    rmm::device_uvector<size_t> sorted_end_row_indices(c_info.size(), stream);
    rmm::device_uvector<size_t> indices(c_info.size(), stream);
    rmm::device_uvector<size_t> sort_order(c_info.size(), stream);

    thrust::sequence(rmm::exec_policy_nosync(stream), indices.begin(), indices.end(), 0);
    thrust::transform(rmm::exec_policy_nosync(stream),
                      c_info.begin(),
                      c_info.end(),
                      end_row_indices.begin(),
                      [] __device__(auto const& c) { return c.end_row_index; });

    auto tmp_bytes = std::size_t{0};
    hipcub::DeviceRadixSort::SortPairs(nullptr,
                                    tmp_bytes,
                                    end_row_indices.begin(),         // keys in
                                    sorted_end_row_indices.begin(),  // sorted keys out
                                    indices.begin(),                 // values in
                                    sort_order.begin(),              // sorted values out
                                    c_info.size(),
                                    0,
                                    sizeof(size_t) * 8,
                                    stream.value());
    auto tmp_stg = rmm::device_buffer(tmp_bytes, stream);
    hipcub::DeviceRadixSort::SortPairs(tmp_stg.data(),
                                    tmp_bytes,
                                    end_row_indices.begin(),         // keys in
                                    sorted_end_row_indices.begin(),  // sorted keys out
                                    indices.begin(),                 // values in
                                    sort_order.begin(),              // sorted values out
                                    c_info.size(),
                                    0,
                                    sizeof(size_t) * 8,
                                    stream.value());

    thrust::transform(rmm::exec_policy_nosync(stream),
                      sort_order.begin(),
                      sort_order.end(),
                      c_info_sorted.begin(),
                      [c_info] __device__(std::size_t i) { return c_info[i]; });
  }

  // page keys grouped by split.
  rmm::device_uvector<int32_t> page_keys_by_split{c_info.size(), stream};
  thrust::transform(rmm::exec_policy_nosync(stream),
                    c_info_sorted.begin(),
                    c_info_sorted.end(),
                    page_keys_by_split.begin(),
                    cuda::proclaim_return_type<int>(
                      [] __device__(cumulative_page_info const& c) { return c.key; }));

  // generate key offsets (offsets to the start of each partition of keys). worst case is 1 page per
  // key
  rmm::device_uvector<size_type> key_offsets(pages.size() + 1, stream);
  auto page_keys             = make_page_key_iterator(pages);
  auto const key_offsets_end = thrust::reduce_by_key(rmm::exec_policy(stream),
                                                     page_keys,
                                                     page_keys + pages.size(),
                                                     thrust::make_constant_iterator(1),
                                                     thrust::make_discard_iterator(),
                                                     key_offsets.begin())
                                 .second;

  size_t const num_unique_keys = key_offsets_end - key_offsets.begin();
  thrust::exclusive_scan(
    rmm::exec_policy_nosync(stream), key_offsets.begin(), key_offsets.end(), key_offsets.begin());

  // adjust the cumulative info such that for each row count, the size includes any pages that span
  // that row count. this is so that if we have this case:
  //              page row counts
  // Column A:    0 <----> 100 <----> 200
  // Column B:    0 <---------------> 200 <--------> 400
  //                        |
  // if we decide to split at row 100, we don't really know the actual amount of bytes in column B
  // at that point.  So we have to proceed as if we are taking the bytes from all 200 rows of that
  // page.
  //
  rmm::device_uvector<cumulative_page_info> aggregated_info(c_info.size(), stream);
  thrust::transform(rmm::exec_policy_nosync(stream),
                    c_info_sorted.begin(),
                    c_info_sorted.end(),
                    aggregated_info.begin(),
                    page_total_size{c_info.data(), key_offsets.data(), num_unique_keys});
  return {std::move(aggregated_info), std::move(page_keys_by_split)};
}

std::tuple<rmm::device_uvector<page_span>, size_t, size_t> compute_next_subpass(
  device_span<cumulative_page_info const> c_info,
  device_span<PageInfo const> pages,
  device_span<ColumnChunkDesc const> chunks,
  device_span<size_type const> page_offsets,
  size_t start_row,
  size_t size_limit,
  size_t num_columns,
  bool is_first_subpass,
  bool has_page_index,
  rmm::cuda_stream_view stream)
{
  auto [aggregated_info, page_keys_by_split] = adjust_cumulative_sizes(c_info, pages, stream);

  // bring back to the cpu
  auto const h_aggregated_info = cudf::detail::make_host_vector(aggregated_info, stream);

#if defined(CHUNKING_DEBUG)
  print_cumulative_page_info(h_aggregated_info, "adjusted");
#endif  // CHUNKING_DEBUG

  // TODO: if the user has explicitly specified skip_rows/num_rows we could be more intelligent
  // about skipping subpasses/pages that do not fall within the range of values, but only if the
  // data does not contain lists (because our row counts are only estimates in that case)

  // find the next split
  auto const start_index = find_start_index(h_aggregated_info, start_row);
  auto const cumulative_size =
    start_row == 0 || start_index == 0 ? 0 : h_aggregated_info[start_index - 1].size_bytes;
  // when choosing subpasses, we need to guarantee at least 2 rows in the included pages so that all
  // list columns have a clear start and end.
  auto const end_index =
    find_next_split(start_index, start_row, cumulative_size, h_aggregated_info, size_limit, 2);
  auto const end_row = h_aggregated_info[end_index].end_row_index;

  // for each column, collect the set of pages that spans start_row / end_row
  rmm::device_uvector<page_span> page_bounds(num_columns, stream);
  auto iter = thrust::make_counting_iterator(size_t{0});
  auto page_row_index =
    cudf::detail::make_counting_transform_iterator(0, get_page_end_row_index{c_info});
  thrust::transform(
    rmm::exec_policy_nosync(stream),
    iter,
    iter + num_columns,
    page_bounds.begin(),
    get_page_span{
      page_offsets, chunks, page_row_index, start_row, end_row, is_first_subpass, has_page_index});

  // total page count over all columns
  auto page_count_iter = thrust::make_transform_iterator(page_bounds.begin(), get_span_size{});
  size_t const total_pages =
    thrust::reduce(rmm::exec_policy(stream), page_count_iter, page_count_iter + num_columns);

  return {
    std::move(page_bounds), total_pages, h_aggregated_info[end_index].size_bytes - cumulative_size};
}

std::vector<row_range> compute_page_splits_by_row(device_span<cumulative_page_info const> c_info,
                                                  device_span<PageInfo const> pages,
                                                  size_t skip_rows,
                                                  size_t num_rows,
                                                  size_t size_limit,
                                                  rmm::cuda_stream_view stream)
{
  auto [aggregated_info, page_keys_by_split] = adjust_cumulative_sizes(c_info, pages, stream);

  // bring back to the cpu
  auto const h_aggregated_info = cudf::detail::make_host_vector(aggregated_info, stream);

#if defined(CHUNKING_DEBUG)
  print_cumulative_page_info(h_aggregated_info, "adjusted");
#endif  // CHUNKING_DEBUG

  std::vector<row_range> splits;
  // note: we are working with absolute row indices so skip_rows represents the absolute min row
  // index we care about
  size_t cur_pos             = find_start_index(h_aggregated_info, skip_rows);
  size_t cur_row_index       = skip_rows;
  size_t cur_cumulative_size = 0;
  auto const max_row = std::min(skip_rows + num_rows, h_aggregated_info.back().end_row_index);
  while (cur_row_index < max_row) {
    auto const split_pos = find_next_split(
      cur_pos, cur_row_index, cur_cumulative_size, h_aggregated_info, size_limit, 1);

    auto const start_row = cur_row_index;
    cur_row_index        = std::min(max_row, h_aggregated_info[split_pos].end_row_index);
    splits.push_back({start_row, cur_row_index - start_row});
    cur_pos             = split_pos;
    cur_cumulative_size = h_aggregated_info[split_pos].size_bytes;
  }

#if defined(CHUNKING_DEBUG)
  print_cumulative_page_info(h_aggregated_info, "adjusted w/splits", splits);
#endif  // CHUNKING_DEBUG

  return splits;
}

[[nodiscard]] std::pair<rmm::device_buffer, rmm::device_buffer> decompress_page_data(
  host_span<ColumnChunkDesc const> chunks,
  host_span<PageInfo> pass_pages,
  host_span<PageInfo> subpass_pages,
  host_span<bool const> subpass_page_mask,
  rmm::cuda_stream_view stream,
  rmm::device_async_resource_ref mr)
{
  CUDF_FUNC_RANGE();

  CUDF_EXPECTS(
    subpass_page_mask.empty() or subpass_page_mask.size() == subpass_pages.size(),
    "Subpass page mask must either be empty or have size equal to the number of subpass pages",
    std::invalid_argument);

  std::array codecs{codec_stats{Compression::BROTLI},
                    codec_stats{Compression::GZIP},
                    codec_stats{Compression::LZ4_RAW},
                    codec_stats{Compression::SNAPPY},
                    codec_stats{Compression::ZSTD}};

  auto is_codec_supported = [&codecs](Compression codec) {
    if (codec == Compression::UNCOMPRESSED) return true;
    return std::find_if(codecs.begin(), codecs.end(), [codec](auto& cstats) {
             return codec == cstats.compression_type;
           }) != codecs.end();
  };

  for (auto const& chunk : chunks) {
    CUDF_EXPECTS(is_codec_supported(chunk.codec),
                 "Unsupported Parquet compression type: " + parquet_compression_name(chunk.codec));
  }

  size_t total_pass_decomp_size = 0;
  for (auto& codec : codecs) {
    // Use an empty span as pass page mask as we don't want to filter out dictionary pages
    codec.add_pages(chunks, pass_pages, codec_stats::page_selection::DICT_PAGES, {});
    total_pass_decomp_size += codec.total_decomp_size;
  }

  // Total number of pages to decompress, including both pass and subpass pages
  size_t num_comp_pages    = 0;
  size_t total_decomp_size = 0;
  for (auto& codec : codecs) {
    codec.add_pages(
      chunks, subpass_pages, codec_stats::page_selection::NON_DICT_PAGES, subpass_page_mask);
    // at this point, the codec contains info for both dictionary pass pages and data subpass pages
    total_decomp_size += codec.total_decomp_size;
    num_comp_pages += codec.num_pages;
  }

  // Dispatch batches of pages to decompress for each codec.
  // Buffer needs to be padded, required by `gpuDecodePageData`.
  rmm::device_buffer pass_decomp_pages(
    cudf::util::round_up_safe(total_pass_decomp_size, cudf::io::detail::BUFFER_PADDING_MULTIPLE),
    stream,
    mr);
  auto const total_subpass_decomp_size = total_decomp_size - total_pass_decomp_size;
  rmm::device_buffer subpass_decomp_pages(
    cudf::util::round_up_safe(total_subpass_decomp_size, cudf::io::detail::BUFFER_PADDING_MULTIPLE),
    stream,
    mr);

  auto comp_in =
    cudf::detail::make_empty_host_vector<device_span<uint8_t const>>(num_comp_pages, stream);
  auto comp_out =
    cudf::detail::make_empty_host_vector<device_span<uint8_t>>(num_comp_pages, stream);

  // vectors to save v2 def and rep level data, if any
  auto copy_in =
    cudf::detail::make_empty_host_vector<device_span<uint8_t const>>(num_comp_pages, stream);
  auto copy_out =
    cudf::detail::make_empty_host_vector<device_span<uint8_t>>(num_comp_pages, stream);

  auto set_parameters = [&](codec_stats& codec,
                            host_span<PageInfo> pages,
                            host_span<bool const> page_mask,
                            void* decomp_data,
                            bool select_dict_pages,
                            size_t& decomp_offset) {
    // Create a page mask iterator that defaults to true if the page_mask is empty
    auto page_mask_iter =
      page_mask.empty() ? thrust::make_constant_iterator(true) : page_mask.begin();

    for (auto page_idx = 0; std::cmp_less(page_idx, pages.size()); ++page_idx) {
      auto& page                = pages[page_idx];
      auto const is_page_needed = page_mask_iter[page_idx];
      // If this is a V2 page, use the `is_compressed` field to determine if it's compressed.
      // For V1 pages, it's always compressed if the chunk.codec is specified.
      auto const is_page_compressed = (page.flags & PAGEINFO_FLAGS_V2) ? page.is_compressed : true;
      if (is_page_needed && chunks[page.chunk_idx].codec == codec.compression_type &&
          (page.flags & PAGEINFO_FLAGS_DICTIONARY) == select_dict_pages and is_page_compressed) {
        auto const dst_base = static_cast<uint8_t*>(decomp_data) + decomp_offset;
        // offset will only be non-zero for V2 pages
        auto const offset =
          page.lvl_bytes[level_type::DEFINITION] + page.lvl_bytes[level_type::REPETITION];
        // for V2 need to copy def and rep level info into place, and then offset the
        // input and output buffers. otherwise we'd have to keep both the compressed
        // and decompressed data.
        if (offset != 0) {
          copy_in.push_back({page.page_data, static_cast<size_t>(offset)});
          copy_out.push_back({dst_base, static_cast<size_t>(offset)});
        }
        // Only decompress if the page contains data after the def/rep levels
        if (page.compressed_page_size > offset) {
          comp_in.push_back(
            {page.page_data + offset, static_cast<size_t>(page.compressed_page_size - offset)});
          comp_out.push_back(
            {dst_base + offset, static_cast<size_t>(page.uncompressed_page_size - offset)});
        } else {
          // If the page wasn't included in the decompression parameters, we need to adjust the
          // page count to allocate results and perform decompression correctly
          --codec.num_pages;
          --num_comp_pages;
        }
        page.page_data = dst_base;
        decomp_offset += page.uncompressed_page_size;
      }
    }
  };

  size_t pass_decomp_offset    = 0;
  size_t subpass_decomp_offset = 0;
  for (auto& codec : codecs) {
    if (codec.num_pages == 0) { continue; }
    // Use empty span as pass page mask as we don't want to filter out dictionary pages
    set_parameters(codec, pass_pages, {}, pass_decomp_pages.data(), true, pass_decomp_offset);
    set_parameters(codec,
                   subpass_pages,
                   subpass_page_mask,
                   subpass_decomp_pages.data(),
                   false,
                   subpass_decomp_offset);
  }

  auto const d_comp_in = cudf::detail::make_device_uvector_async(
    comp_in, stream, cudf::get_current_device_resource_ref());
  auto const d_comp_out = cudf::detail::make_device_uvector_async(
    comp_out, stream, cudf::get_current_device_resource_ref());
  rmm::device_uvector<codec_exec_result> comp_res(num_comp_pages, stream);
  thrust::uninitialized_fill(rmm::exec_policy_nosync(stream),
                             comp_res.begin(),
                             comp_res.end(),
                             codec_exec_result{0, codec_status::FAILURE});

  int32_t start_pos = 0;
  for (auto const& codec : codecs) {
    if (codec.num_pages == 0) { continue; }
    CUDF_EXPECTS(is_supported_read_parquet(from_parquet_compression(codec.compression_type)),
                 "Unsupported compression type for Parquet reading");

    device_span<device_span<uint8_t const> const> d_comp_in_view{d_comp_in.data() + start_pos,
                                                                 codec.num_pages};
    device_span<device_span<uint8_t> const> d_comp_out_view(d_comp_out.data() + start_pos,
                                                            codec.num_pages);
    device_span<codec_exec_result> d_comp_res_view(comp_res.data() + start_pos, codec.num_pages);
    cudf::io::detail::decompress(from_parquet_compression(codec.compression_type),
                                 d_comp_in_view,
                                 d_comp_out_view,
                                 d_comp_res_view,
                                 codec.max_decompressed_size,
                                 codec.total_decomp_size,
                                 stream);

    start_pos += codec.num_pages;
  }
  // now copy the uncompressed V2 def and rep level data
  if (not copy_in.empty()) {
    auto const d_copy_in = cudf::detail::make_device_uvector_async(
      copy_in, stream, cudf::get_current_device_resource_ref());
    auto const d_copy_out = cudf::detail::make_device_uvector_async(
      copy_out, stream, cudf::get_current_device_resource_ref());

    cudf::io::detail::gpu_copy_uncompressed_blocks(d_copy_in, d_copy_out, stream);
  }

  CUDF_EXPECTS(
    thrust::all_of(rmm::exec_policy(stream),
                   comp_res.begin(),
                   comp_res.end(),
                   [] __device__(auto const& res) { return res.status == codec_status::SUCCESS; }),
    "Error during decompression");

  return {std::move(pass_decomp_pages), std::move(subpass_decomp_pages)};
}

void detect_malformed_pages(device_span<PageInfo const> pages,
                            device_span<ColumnChunkDesc const> chunks,
                            std::optional<size_t> expected_row_count,
                            rmm::cuda_stream_view stream)
{
  CUDF_FUNC_RANGE();

  // sum row counts for all non-dictionary, non-list columns. other columns will be indicated as 0
  rmm::device_uvector<size_type> row_counts(pages.size(),
                                            stream);  // worst case:  num keys == num pages
  auto const size_iter =
    thrust::make_transform_iterator(pages.begin(), flat_column_num_rows{chunks.data()});
  auto const row_counts_begin = row_counts.begin();
  auto page_keys              = make_page_key_iterator(pages);
  auto const row_counts_end   = thrust::reduce_by_key(rmm::exec_policy(stream),
                                                    page_keys,
                                                    page_keys + pages.size(),
                                                    size_iter,
                                                    thrust::make_discard_iterator(),
                                                    row_counts_begin)
                                .second;

  // make sure all non-zero row counts are the same
  rmm::device_uvector<size_type> compacted_row_counts(pages.size(), stream);
  auto const compacted_row_counts_begin = compacted_row_counts.begin();
  auto const compacted_row_counts_end   = thrust::copy_if(rmm::exec_policy(stream),
                                                        row_counts_begin,
                                                        row_counts_end,
                                                        compacted_row_counts_begin,
                                                        row_counts_nonzero{});
  if (compacted_row_counts_end != compacted_row_counts_begin) {
    auto const found_row_count = static_cast<size_t>(compacted_row_counts.element(0, stream));

    // if we somehow don't match the expected row count from the row groups themselves
    if (expected_row_count.has_value()) {
      CUDF_EXPECTS(expected_row_count.value() == found_row_count,
                   "Encountered malformed parquet page data (unexpected row count in page data)");
    }

    // all non-zero row counts must be the same
    auto const chk =
      thrust::count_if(rmm::exec_policy(stream),
                       compacted_row_counts_begin,
                       compacted_row_counts_end,
                       row_counts_different{static_cast<size_type>(found_row_count)});
    CUDF_EXPECTS(chk == 0,
                 "Encountered malformed parquet page data (row count mismatch in page data)");
  }
}

rmm::device_uvector<size_t> compute_decompression_scratch_sizes(
  device_span<ColumnChunkDesc const> chunks,
  device_span<PageInfo const> pages,
  rmm::cuda_stream_view stream)
{
  auto page_keys = make_page_key_iterator(pages);

  // per-codec page counts and decompression sizes
  rmm::device_uvector<decompression_info> decomp_info(pages.size(), stream);
  auto decomp_iter = thrust::make_transform_iterator(pages.begin(), get_decomp_info{chunks});
  thrust::inclusive_scan_by_key(rmm::exec_policy_nosync(stream),
                                page_keys,
                                page_keys + pages.size(),
                                decomp_iter,
                                decomp_info.begin(),
                                cuda::std::equal_to<int32_t>{},
                                decomp_sum{});

  // retrieve to host so we can get compression scratch sizes
  auto h_decomp_info = cudf::detail::make_host_vector(decomp_info, stream);
  auto temp_cost     = cudf::detail::make_host_vector<size_t>(pages.size(), stream);
  std::transform(h_decomp_info.begin(), h_decomp_info.end(), temp_cost.begin(), [](auto const& d) {
    return cudf::io::detail::get_decompression_scratch_size(d);
  });

  rmm::device_uvector<size_t> d_temp_cost = cudf::detail::make_device_uvector_async(
    temp_cost, stream, cudf::get_current_device_resource_ref());

  std::array codecs{compression_type::BROTLI,
                    compression_type::GZIP,
                    compression_type::LZ4,
                    compression_type::SNAPPY,
                    compression_type::ZSTD};
  for (auto const codec : codecs) {
    if (cudf::io::detail::is_decompression_scratch_size_ex_supported(codec)) {
      auto const total_decomp_info = thrust::transform_reduce(
        rmm::exec_policy(stream),
        decomp_iter,
        decomp_iter + pages.size(),
        cuda::proclaim_return_type<decompression_info>(
          [codec] __device__(decompression_info const& d) {
            return d.type == codec ? d : decompression_info{codec, 0, 0, 0};
          }),
        decompression_info{codec, 0, 0, 0},
        decomp_sum{});

      // Collect pages with matching codecs
      rmm::device_uvector<device_span<uint8_t const>> temp_spans(pages.size(), stream);
      auto iter = thrust::make_counting_iterator(size_t{0});
      thrust::for_each(
        rmm::exec_policy_nosync(stream),
        iter,
        iter + pages.size(),
        [pages      = pages.begin(),
         chunks     = chunks.begin(),
         temp_spans = temp_spans.begin(),
         codec] __device__(size_t i) {
          auto const& page = pages[i];
          if (parquet_compression_support(chunks[page.chunk_idx].codec).first == codec) {
            temp_spans[i] = {page.page_data, static_cast<size_t>(page.compressed_page_size)};
          } else {
            temp_spans[i] = {nullptr, 0};  // Mark pages with other codecs as empty
          }
        });
      // Copy only non-null spans
      rmm::device_uvector<device_span<uint8_t const>> page_spans(pages.size(), stream);
      auto end_iter =
        thrust::copy_if(rmm::exec_policy_nosync(stream),
                        temp_spans.begin(),
                        temp_spans.end(),
                        page_spans.begin(),
                        [] __device__(auto const& span) { return span.data() != nullptr; });
      if (end_iter == page_spans.begin()) {
        // No pages compressed with this codec, skip
        continue;
      }
      page_spans.resize(end_iter - page_spans.begin(), stream);

      auto const total_temp_size    = get_decompression_scratch_size(total_decomp_info);
      auto const total_temp_size_ex = cudf::io::detail::get_decompression_scratch_size_ex(
        total_decomp_info.type,
        page_spans,
        total_decomp_info.max_page_decompressed_size,
        total_decomp_info.total_decompressed_size,
        stream);

      // Make use of the extended API if it provides a more accurate estimate
      if (total_temp_size_ex < total_temp_size) {
        // The new extended API provides a more accurate (smaller) estimate than the legacy API.
        // We cannot efficiently use the extended API to get per-page scratch sizes, so we adjust
        // the per-page scratch sizes to on-average reflect the better estimate. This means that
        // the scratch size might not be accurate for each page, but it will in aggregate.
        auto const adjustment_ratio = static_cast<double>(total_temp_size_ex) / total_temp_size;

        // Apply the adjustment ratio to each page's temporary cost
        thrust::for_each(rmm::exec_policy_nosync(stream),
                         thrust::make_counting_iterator(size_t{0}),
                         thrust::make_counting_iterator(pages.size()),
                         [pages           = pages.begin(),
                          chunks          = chunks.begin(),
                          d_temp_cost_ptr = d_temp_cost.begin(),
                          adjustment_ratio,
                          codec] __device__(size_t i) {
                           auto const page_codec =
                             parquet_compression_support(chunks[pages[i].chunk_idx].codec).first;
                           // Only adjust pages that use the current compression codec
                           if (page_codec == codec) {
                             auto const cost = d_temp_cost_ptr[i];
                             // Scale down the cost and round up to ensure we don't underestimate
                             auto const adjusted =
                               static_cast<size_t>(cuda::std::ceil(cost * adjustment_ratio));
                             d_temp_cost_ptr[i] = adjusted;
                           }
                         });
      }
    }
  }
  return d_temp_cost;
}

void include_decompression_scratch_size(device_span<size_t const> temp_cost,
                                        device_span<cumulative_page_info> c_info,
                                        rmm::cuda_stream_view stream)
{
  auto iter = thrust::make_counting_iterator(size_t{0});
  thrust::for_each(rmm::exec_policy_nosync(stream),
                   iter,
                   iter + c_info.size(),
                   [temp_cost = temp_cost.begin(), c_info = c_info.begin()] __device__(size_t i) {
                     c_info[i].size_bytes += temp_cost[i];
                   });
}

}  // namespace cudf::io::parquet::detail
