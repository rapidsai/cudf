#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019-2021, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

/**
 * @file writer_impl.cu
 * @brief cuDF-IO parquet writer class implementation
 */

#include <io/statistics/column_statistics.cuh>
#include "writer_impl.hpp"

#include <io/utilities/column_utils.cuh>
#include "compact_protocol_writer.hpp"

#include <cudf/column/column_device_view.cuh>
#include <cudf/detail/utilities/vector_factories.hpp>
#include <cudf/lists/lists_column_view.hpp>
#include <cudf/null_mask.hpp>
#include <cudf/strings/strings_column_view.hpp>
#include <cudf/table/table_device_view.cuh>

#include <rmm/cuda_stream_view.hpp>
#include <rmm/device_buffer.hpp>
#include <rmm/device_scalar.hpp>
#include <rmm/device_uvector.hpp>

#include <algorithm>
#include <cstring>
#include <numeric>
#include <utility>

namespace cudf {
namespace io {
namespace detail {
namespace parquet {
using namespace cudf::io::parquet;
using namespace cudf::io;

namespace {
/**
 * @brief Helper for pinned host memory
 */
template <typename T>
using pinned_buffer = std::unique_ptr<T, decltype(&hipHostFree)>;

/**
 * @brief Function that translates GDF compression to parquet compression
 */
parquet::Compression to_parquet_compression(compression_type compression)
{
  switch (compression) {
    case compression_type::AUTO:
    case compression_type::SNAPPY: return parquet::Compression::SNAPPY;
    case compression_type::NONE: return parquet::Compression::UNCOMPRESSED;
    default:
      CUDF_EXPECTS(false, "Unsupported compression type");
      return parquet::Compression::UNCOMPRESSED;
  }
}

}  // namespace

struct linked_column_view;

using LinkedColPtr    = std::shared_ptr<linked_column_view>;
using LinkedColVector = std::vector<LinkedColPtr>;

/**
 * @brief column_view with the added member pointer to the parent of this column.
 *
 */
struct linked_column_view : public column_view {
  // TODO(cp): we are currently keeping all column_view children info multiple times - once for each
  //       copy of this object. Options:
  // 1. Inherit from column_view_base. Only lose out on children vector. That is not needed.
  // 2. Don't inherit at all. make linked_column_view keep a reference wrapper to its column_view
  linked_column_view(column_view const& col) : column_view(col), parent(nullptr)
  {
    for (auto child_it = col.child_begin(); child_it < col.child_end(); ++child_it) {
      children.push_back(std::make_shared<linked_column_view>(this, *child_it));
    }
  }

  linked_column_view(linked_column_view* parent, column_view const& col)
    : column_view(col), parent(parent)
  {
    for (auto child_it = col.child_begin(); child_it < col.child_end(); ++child_it) {
      children.push_back(std::make_shared<linked_column_view>(this, *child_it));
    }
  }

  linked_column_view* parent;  //!< Pointer to parent of this column. Nullptr if root
  LinkedColVector children;
};

/**
 * @brief Converts all column_views of a table into linked_column_views
 *
 * @param table table of columns to convert
 * @return Vector of converted linked_column_views
 */
LinkedColVector input_table_to_linked_columns(table_view const& table)
{
  LinkedColVector result;
  for (column_view const& col : table) {
    result.emplace_back(std::make_shared<linked_column_view>(col));
  }

  return result;
}

/**
 * @brief Extends SchemaElement to add members required in constructing parquet_column_view
 *
 * Added members are:
 * 1. leaf_column: Pointer to leaf linked_column_view which points to the corresponding data stream
 *    of a leaf schema node. For non-leaf struct node, this is nullptr.
 * 2. stats_dtype: datatype for statistics calculation required for the data stream of a leaf node.
 * 3. ts_scale: scale to multiply or divide timestamp by in order to convert timestamp to parquet
 *    supported types
 */
struct schema_tree_node : public SchemaElement {
  LinkedColPtr leaf_column;
  statistics_dtype stats_dtype;
  int32_t ts_scale;

  // TODO(fut): Think about making schema a class that holds a vector of schema_tree_nodes. The
  // function construct_schema_tree could be its constructor. It can have method to get the per
  // column nullability given a schema node index corresponding to a leaf schema. Much easier than
  // that is a method to get path in schema, given a leaf node
};

struct leaf_schema_fn {
  schema_tree_node& col_schema;
  LinkedColPtr const& col;
  column_in_metadata const& col_meta;
  bool timestamp_is_int96;

  template <typename T>
  std::enable_if_t<std::is_same<T, bool>::value, void> operator()()
  {
    col_schema.type        = Type::BOOLEAN;
    col_schema.stats_dtype = statistics_dtype::dtype_bool;
  }

  template <typename T>
  std::enable_if_t<std::is_same<T, int8_t>::value, void> operator()()
  {
    col_schema.type           = Type::INT32;
    col_schema.converted_type = ConvertedType::INT_8;
    col_schema.stats_dtype    = statistics_dtype::dtype_int8;
  }

  template <typename T>
  std::enable_if_t<std::is_same<T, int16_t>::value, void> operator()()
  {
    col_schema.type           = Type::INT32;
    col_schema.converted_type = ConvertedType::INT_16;
    col_schema.stats_dtype    = statistics_dtype::dtype_int16;
  }

  template <typename T>
  std::enable_if_t<std::is_same<T, int32_t>::value, void> operator()()
  {
    col_schema.type        = Type::INT32;
    col_schema.stats_dtype = statistics_dtype::dtype_int32;
  }

  template <typename T>
  std::enable_if_t<std::is_same<T, int64_t>::value, void> operator()()
  {
    col_schema.type        = Type::INT64;
    col_schema.stats_dtype = statistics_dtype::dtype_int64;
  }

  template <typename T>
  std::enable_if_t<std::is_same<T, uint8_t>::value, void> operator()()
  {
    col_schema.type           = Type::INT32;
    col_schema.converted_type = ConvertedType::UINT_8;
    col_schema.stats_dtype    = statistics_dtype::dtype_int8;
  }

  template <typename T>
  std::enable_if_t<std::is_same<T, uint16_t>::value, void> operator()()
  {
    col_schema.type           = Type::INT32;
    col_schema.converted_type = ConvertedType::UINT_16;
    col_schema.stats_dtype    = statistics_dtype::dtype_int16;
  }

  template <typename T>
  std::enable_if_t<std::is_same<T, uint32_t>::value, void> operator()()
  {
    col_schema.type           = Type::INT32;
    col_schema.converted_type = ConvertedType::UINT_32;
    col_schema.stats_dtype    = statistics_dtype::dtype_int32;
  }

  template <typename T>
  std::enable_if_t<std::is_same<T, uint64_t>::value, void> operator()()
  {
    col_schema.type           = Type::INT64;
    col_schema.converted_type = ConvertedType::UINT_64;
    col_schema.stats_dtype    = statistics_dtype::dtype_int64;
  }

  template <typename T>
  std::enable_if_t<std::is_same<T, float>::value, void> operator()()
  {
    col_schema.type        = Type::FLOAT;
    col_schema.stats_dtype = statistics_dtype::dtype_float32;
  }

  template <typename T>
  std::enable_if_t<std::is_same<T, double>::value, void> operator()()
  {
    col_schema.type        = Type::DOUBLE;
    col_schema.stats_dtype = statistics_dtype::dtype_float64;
  }

  template <typename T>
  std::enable_if_t<std::is_same<T, cudf::string_view>::value, void> operator()()
  {
    col_schema.type           = Type::BYTE_ARRAY;
    col_schema.converted_type = ConvertedType::UTF8;
    col_schema.stats_dtype    = statistics_dtype::dtype_string;
  }

  template <typename T>
  std::enable_if_t<std::is_same<T, cudf::timestamp_D>::value, void> operator()()
  {
    col_schema.type           = Type::INT32;
    col_schema.converted_type = ConvertedType::DATE;
    col_schema.stats_dtype    = statistics_dtype::dtype_int32;
  }

  template <typename T>
  std::enable_if_t<std::is_same<T, cudf::timestamp_s>::value, void> operator()()
  {
    col_schema.type = (timestamp_is_int96) ? Type::INT96 : Type::INT64;
    col_schema.converted_type =
      (timestamp_is_int96) ? ConvertedType::UNKNOWN : ConvertedType::TIMESTAMP_MILLIS;
    col_schema.stats_dtype = statistics_dtype::dtype_timestamp64;
    col_schema.ts_scale    = 1000;
  }

  template <typename T>
  std::enable_if_t<std::is_same<T, cudf::timestamp_ms>::value, void> operator()()
  {
    col_schema.type = (timestamp_is_int96) ? Type::INT96 : Type::INT64;
    col_schema.converted_type =
      (timestamp_is_int96) ? ConvertedType::UNKNOWN : ConvertedType::TIMESTAMP_MILLIS;
    col_schema.stats_dtype = statistics_dtype::dtype_timestamp64;
  }

  template <typename T>
  std::enable_if_t<std::is_same<T, cudf::timestamp_us>::value, void> operator()()
  {
    col_schema.type = (timestamp_is_int96) ? Type::INT96 : Type::INT64;
    col_schema.converted_type =
      (timestamp_is_int96) ? ConvertedType::UNKNOWN : ConvertedType::TIMESTAMP_MICROS;
    col_schema.stats_dtype = statistics_dtype::dtype_timestamp64;
  }

  template <typename T>
  std::enable_if_t<std::is_same<T, cudf::timestamp_ns>::value, void> operator()()
  {
    col_schema.type = (timestamp_is_int96) ? Type::INT96 : Type::INT64;
    col_schema.converted_type =
      (timestamp_is_int96) ? ConvertedType::UNKNOWN : ConvertedType::TIMESTAMP_MICROS;
    col_schema.stats_dtype = statistics_dtype::dtype_timestamp64;
    col_schema.ts_scale    = -1000;  // negative value indicates division by absolute value
  }

  //  unsupported outside cudf for parquet 1.0.
  template <typename T>
  std::enable_if_t<std::is_same<T, cudf::duration_D>::value, void> operator()()
  {
    col_schema.type           = Type::INT32;
    col_schema.converted_type = ConvertedType::TIME_MILLIS;
    col_schema.stats_dtype    = statistics_dtype::dtype_int64;
  }

  template <typename T>
  std::enable_if_t<std::is_same<T, cudf::duration_s>::value, void> operator()()
  {
    col_schema.type           = Type::INT64;
    col_schema.converted_type = ConvertedType::TIME_MILLIS;
    col_schema.stats_dtype    = statistics_dtype::dtype_int64;
    col_schema.ts_scale       = 1000;
  }

  template <typename T>
  std::enable_if_t<std::is_same<T, cudf::duration_ms>::value, void> operator()()
  {
    col_schema.type           = Type::INT64;
    col_schema.converted_type = ConvertedType::TIME_MILLIS;
    col_schema.stats_dtype    = statistics_dtype::dtype_int64;
  }

  template <typename T>
  std::enable_if_t<std::is_same<T, cudf::duration_us>::value, void> operator()()
  {
    col_schema.type           = Type::INT64;
    col_schema.converted_type = ConvertedType::TIME_MICROS;
    col_schema.stats_dtype    = statistics_dtype::dtype_int64;
  }

  //  unsupported outside cudf for parquet 1.0.
  template <typename T>
  std::enable_if_t<std::is_same<T, cudf::duration_ns>::value, void> operator()()
  {
    col_schema.type           = Type::INT64;
    col_schema.converted_type = ConvertedType::TIME_MICROS;
    col_schema.stats_dtype    = statistics_dtype::dtype_int64;
    col_schema.ts_scale       = -1000;  // negative value indicates division by absolute value
  }

  template <typename T>
  std::enable_if_t<cudf::is_fixed_point<T>(), void> operator()()
  {
    if (std::is_same<T, numeric::decimal32>::value) {
      col_schema.type        = Type::INT32;
      col_schema.stats_dtype = statistics_dtype::dtype_int32;
    } else if (std::is_same<T, numeric::decimal64>::value) {
      col_schema.type        = Type::INT64;
      col_schema.stats_dtype = statistics_dtype::dtype_decimal64;
    } else {
      CUDF_FAIL("Unsupported fixed point type for parquet writer");
    }
    col_schema.converted_type = ConvertedType::DECIMAL;
    col_schema.decimal_scale = -col->type().scale();  // parquet and cudf disagree about scale signs
    CUDF_EXPECTS(col_meta.is_decimal_precision_set(),
                 "Precision must be specified for decimal columns");
    CUDF_EXPECTS(col_meta.get_decimal_precision() >= col_schema.decimal_scale,
                 "Precision must be equal to or greater than scale!");
    col_schema.decimal_precision = col_meta.get_decimal_precision();
  }

  template <typename T>
  std::enable_if_t<cudf::is_nested<T>(), void> operator()()
  {
    CUDF_FAIL("This functor is only meant for physical data types");
  }

  template <typename T>
  std::enable_if_t<cudf::is_dictionary<T>(), void> operator()()
  {
    CUDF_FAIL("Dictionary columns are not supported for writing");
  }
};

/**
 * @brief Construct schema from input columns and per-column input options
 *
 * Recursively traverses through linked_columns and corresponding metadata to construct schema tree.
 * The resulting schema tree is stored in a vector in pre-order traversal order.
 */
std::vector<schema_tree_node> construct_schema_tree(LinkedColVector const& linked_columns,
                                                    table_input_metadata const& metadata,
                                                    bool single_write_mode,
                                                    bool int96_timestamps)
{
  std::vector<schema_tree_node> schema;
  schema_tree_node root{};
  root.type            = UNDEFINED_TYPE;
  root.repetition_type = NO_REPETITION_TYPE;
  root.name            = "schema";
  root.num_children    = linked_columns.size();
  root.parent_idx      = -1;  // root schema has no parent
  schema.push_back(std::move(root));

  std::function<void(LinkedColPtr const&, column_in_metadata const&, size_t)> add_schema =
    [&](LinkedColPtr const& col, column_in_metadata const& col_meta, size_t parent_idx) {
      bool col_nullable = [&]() {
        if (single_write_mode) {
          return col->nullable();
        } else {
          if (col_meta.is_nullability_defined()) {
            if (col_meta.nullable() == false) {
              CUDF_EXPECTS(
                col->nullable() == false,
                "Mismatch in metadata prescribed nullability and input column nullability. "
                "Metadata for nullable input column cannot prescribe nullability = false");
            }
            return col_meta.nullable();
          } else {
            // For chunked write, when not provided nullability, we assume the worst case scenario
            // that all columns are nullable.
            return true;
          }
        }
      }();

      if (col->type().id() == type_id::STRUCT) {
        // if struct, add current and recursively call for all children
        schema_tree_node struct_schema{};
        struct_schema.repetition_type =
          col_nullable ? FieldRepetitionType::OPTIONAL : FieldRepetitionType::REQUIRED;

        struct_schema.name = (schema[parent_idx].name == "list") ? "element" : col_meta.get_name();
        struct_schema.num_children = col->num_children();
        struct_schema.parent_idx   = parent_idx;
        schema.push_back(std::move(struct_schema));

        auto struct_node_index = schema.size() - 1;
        // for (auto child_it = col->children.begin(); child_it < col->children.end(); child_it++) {
        //   add_schema(*child_it, struct_node_index);
        // }
        CUDF_EXPECTS(col->num_children() == static_cast<int>(col_meta.num_children()),
                     "Mismatch in number of child columns between input table and metadata");
        for (size_t i = 0; i < col->children.size(); ++i) {
          add_schema(col->children[i], col_meta.child(i), struct_node_index);
        }
      } else if (col->type().id() == type_id::LIST) {
        // List schema is denoted by two levels for each nesting level and one final level for leaf.
        // The top level is the same name as the column name.
        // So e.g. List<List<int>> is denoted in the schema by
        // "col_name" : { "list" : { "element" : { "list" : { "element" } } } }

        schema_tree_node list_schema_1{};
        list_schema_1.converted_type = ConvertedType::LIST;
        list_schema_1.repetition_type =
          col_nullable ? FieldRepetitionType::OPTIONAL : FieldRepetitionType::REQUIRED;
        list_schema_1.name = (schema[parent_idx].name == "list") ? "element" : col_meta.get_name();
        list_schema_1.num_children = 1;
        list_schema_1.parent_idx   = parent_idx;
        schema.push_back(std::move(list_schema_1));

        schema_tree_node list_schema_2{};
        list_schema_2.repetition_type = FieldRepetitionType::REPEATED;
        list_schema_2.name            = "list";
        list_schema_2.num_children    = 1;
        list_schema_2.parent_idx      = schema.size() - 1;  // Parent is list_schema_1, last added.
        schema.push_back(std::move(list_schema_2));

        CUDF_EXPECTS(col_meta.num_children() == 2,
                     "List column's metadata should have exactly two children");

        add_schema(col->children[lists_column_view::child_column_index],
                   col_meta.child(lists_column_view::child_column_index),
                   schema.size() - 1);
      } else {
        // if leaf, add current
        if (col->type().id() == type_id::STRING) {
          CUDF_EXPECTS(col_meta.num_children() == 2 or col_meta.num_children() == 0,
                       "String column's corresponding metadata should have zero or two children");
        } else {
          CUDF_EXPECTS(col_meta.num_children() == 0,
                       "Leaf column's corresponding metadata cannot have children");
        }

        schema_tree_node col_schema{};

        bool timestamp_is_int96 = int96_timestamps or col_meta.is_enabled_int96_timestamps();

        cudf::type_dispatcher(col->type(),
                              leaf_schema_fn{col_schema, col, col_meta, timestamp_is_int96});

        col_schema.repetition_type = col_nullable ? OPTIONAL : REQUIRED;
        col_schema.name = (schema[parent_idx].name == "list") ? "element" : col_meta.get_name();
        col_schema.parent_idx  = parent_idx;
        col_schema.leaf_column = col;
        schema.push_back(col_schema);
      }
    };

  CUDF_EXPECTS(metadata.column_metadata.size() == linked_columns.size(),
               "Mismatch in the number of columns and the corresponding metadata elements");
  // Add all linked_columns to schema using parent_idx = 0 (root)
  for (size_t i = 0; i < linked_columns.size(); ++i) {
    add_schema(linked_columns[i], metadata.column_metadata[i], 0);
  }

  return schema;
}

/**
 * @brief Class to store parquet specific information for one data stream.
 *
 * Contains information about a single data stream. In case of struct columns, a data stream is one
 * of the child leaf columns that contains data.
 * e.g. A column Struct<int, List<float>> contains 2 data streams:
 * - Struct<int>
 * - Struct<List<float>>
 *
 */
struct parquet_column_view {
  parquet_column_view(schema_tree_node const& schema_node,
                      std::vector<schema_tree_node> const& schema_tree,
                      rmm::cuda_stream_view stream);

  column_view leaf_column_view() const;
  gpu::parquet_column_device_view get_device_view(rmm::cuda_stream_view stream);

  column_view cudf_column_view() const { return cudf_col; }
  parquet::Type physical_type() const { return schema_node.type; }

  std::vector<std::string> const& get_path_in_schema() { return path_in_schema; }

  // LIST related member functions
  uint8_t max_def_level() const noexcept { return _max_def_level; }
  uint8_t max_rep_level() const noexcept { return _max_rep_level; }
  bool is_list() const noexcept { return _is_list; }

  // Dictionary related member functions
  uint32_t* get_dict_data() { return (_dict_data.size()) ? _dict_data.data() : nullptr; }
  uint32_t* get_dict_index() { return (_dict_index.size()) ? _dict_index.data() : nullptr; }
  void use_dictionary(bool use_dict) { _dictionary_used = use_dict; }
  void alloc_dictionary(size_t max_num_rows, rmm::cuda_stream_view stream)
  {
    _dict_data.resize(max_num_rows, stream);
    _dict_index.resize(max_num_rows, stream);
  }
  bool check_dictionary_used(rmm::cuda_stream_view stream)
  {
    if (!_dictionary_used) {
      _dict_data.resize(0, stream);
      _dict_data.shrink_to_fit(stream);
      _dict_index.resize(0, stream);
      _dict_index.shrink_to_fit(stream);
    }
    return _dictionary_used;
  }

 private:
  // Schema related members
  schema_tree_node schema_node;
  std::vector<std::string> path_in_schema;
  uint8_t _max_def_level = 0;
  uint8_t _max_rep_level = 0;
  rmm::device_uvector<uint8_t> _d_nullability;

  column_view cudf_col;

  // List-related members
  bool _is_list;
  rmm::device_uvector<size_type>
    _dremel_offsets;  ///< For each row, the absolute offset into the repetition and definition
                      ///< level vectors. O(num rows)
  rmm::device_uvector<uint8_t> _rep_level;
  rmm::device_uvector<uint8_t> _def_level;
  std::vector<uint8_t> _nullability;
  size_type _data_count = 0;

  // Dictionary related members
  bool _dictionary_used = false;
  rmm::device_uvector<uint32_t> _dict_data;
  rmm::device_uvector<uint32_t> _dict_index;
};

parquet_column_view::parquet_column_view(schema_tree_node const& schema_node,
                                         std::vector<schema_tree_node> const& schema_tree,
                                         rmm::cuda_stream_view stream)
  : schema_node(schema_node),
    _d_nullability(0, stream),
    _dremel_offsets(0, stream),
    _rep_level(0, stream),
    _def_level(0, stream),
    _dict_data(0, stream),
    _dict_index(0, stream)
{
  // Construct single inheritance column_view from linked_column_view
  auto curr_col                           = schema_node.leaf_column.get();
  column_view single_inheritance_cudf_col = *curr_col;
  while (curr_col->parent) {
    auto const& parent = *curr_col->parent;

    // For list columns, we still need to retain the offset child column.
    auto children =
      (parent.type().id() == type_id::LIST)
        ? std::vector<column_view>{parent.child(lists_column_view::offsets_column_index),
                                   single_inheritance_cudf_col}
        : std::vector<column_view>{single_inheritance_cudf_col};

    single_inheritance_cudf_col = column_view(parent.type(),
                                              parent.size(),
                                              parent.head(),
                                              parent.null_mask(),
                                              UNKNOWN_NULL_COUNT,
                                              parent.offset(),
                                              children);

    curr_col = curr_col->parent;
  }
  cudf_col = single_inheritance_cudf_col;

  // Construct path_in_schema by travelling up in the schema_tree
  std::vector<std::string> path;
  auto curr_schema_node = schema_node;
  do {
    path.push_back(curr_schema_node.name);
    if (curr_schema_node.parent_idx != -1) {
      curr_schema_node = schema_tree[curr_schema_node.parent_idx];
    }
  } while (curr_schema_node.parent_idx != -1);
  path_in_schema = std::vector<std::string>(path.crbegin(), path.crend());

  // Calculate max definition level by counting the number of levels that are optional (nullable)
  // and max repetition level by counting the number of REPEATED levels in this column's hierarchy
  uint16_t max_def_level = 0;
  uint16_t max_rep_level = 0;
  curr_schema_node       = schema_node;
  while (curr_schema_node.parent_idx != -1) {
    if (curr_schema_node.repetition_type == parquet::REPEATED or
        curr_schema_node.repetition_type == parquet::OPTIONAL) {
      ++max_def_level;
    }
    if (curr_schema_node.repetition_type == parquet::REPEATED) { ++max_rep_level; }
    curr_schema_node = schema_tree[curr_schema_node.parent_idx];
  }
  CUDF_EXPECTS(max_def_level < 256, "Definition levels above 255 are not supported");
  CUDF_EXPECTS(max_rep_level < 256, "Definition levels above 255 are not supported");

  _max_def_level = max_def_level;
  _max_rep_level = max_rep_level;

  // Construct nullability vector using repetition_type from schema.
  std::vector<uint8_t> r_nullability;
  curr_schema_node = schema_node;
  while (curr_schema_node.parent_idx != -1) {
    if (not curr_schema_node.is_stub()) {
      r_nullability.push_back(curr_schema_node.repetition_type == FieldRepetitionType::OPTIONAL);
    }
    curr_schema_node = schema_tree[curr_schema_node.parent_idx];
  }
  _nullability = std::vector<uint8_t>(r_nullability.crbegin(), r_nullability.crend());
  // TODO(cp): Explore doing this for all columns in a single go outside this ctor. Maybe using
  // hostdevice_vector. Currently this involves a hipMemcpyAsync for each column.
  _d_nullability = rmm::device_uvector<uint8_t>(_nullability.size(), stream);
  CUDA_TRY(hipMemcpyAsync(_d_nullability.data(),
                           _nullability.data(),
                           _nullability.size() * sizeof(uint8_t),
                           hipMemcpyHostToDevice,
                           stream.value()));

  _is_list = (_max_rep_level > 0);

  if (cudf_col.size() == 0) { return; }

  if (_is_list) {
    // Top level column's offsets are not applied to all children. Get the effective offset and
    // size of the leaf column
    // Calculate row offset into dremel data (repetition/definition values) and the respective
    // definition and repetition levels
    gpu::dremel_data dremel = gpu::get_dremel_data(cudf_col, _d_nullability, _nullability, stream);
    _dremel_offsets         = std::move(dremel.dremel_offsets);
    _rep_level              = std::move(dremel.rep_level);
    _def_level              = std::move(dremel.def_level);
    _data_count = dremel.leaf_data_size;  // Needed for knowing what size dictionary to allocate

    stream.synchronize();
  } else {
    // For non-list struct, the size of the root column is the same as the size of the leaf column
    _data_count = cudf_col.size();
  }
}

column_view parquet_column_view::leaf_column_view() const
{
  auto col = cudf_col;
  while (cudf::is_nested(col.type())) {
    if (col.type().id() == type_id::LIST) {
      col = col.child(lists_column_view::child_column_index);
    } else if (col.type().id() == type_id::STRUCT) {
      col = col.child(0);  // Stored cudf_col has only one child if struct
    }
  }
  return col;
}

gpu::parquet_column_device_view parquet_column_view::get_device_view(rmm::cuda_stream_view stream)
{
  column_view col  = leaf_column_view();
  auto desc        = gpu::parquet_column_device_view{};  // Zero out all fields
  desc.stats_dtype = schema_node.stats_dtype;
  desc.ts_scale    = schema_node.ts_scale;

  // TODO (dm): Enable dictionary for list and struct after refactor
  if (physical_type() != BOOLEAN && physical_type() != UNDEFINED_TYPE &&
      !is_nested(cudf_col.type())) {
    alloc_dictionary(_data_count, stream);
    desc.dict_index = get_dict_index();
    desc.dict_data  = get_dict_data();
  }

  if (is_list()) {
    desc.level_offsets = _dremel_offsets.data();
    desc.rep_values    = _rep_level.data();
    desc.def_values    = _def_level.data();
  }
  desc.num_rows      = cudf_col.size();
  desc.physical_type = static_cast<uint8_t>(physical_type());
  auto count_bits    = [](uint16_t number) {
    int16_t nbits = 0;
    while (number > 0) {
      nbits++;
      number >>= 1;
    }
    return nbits;
  };
  desc.level_bits  = count_bits(max_rep_level()) << 4 | count_bits(max_def_level());
  desc.nullability = _d_nullability.data();
  return desc;
}

void writer::impl::init_page_fragments(cudf::detail::hostdevice_2dvector<gpu::PageFragment>& frag,
                                       device_span<gpu::parquet_column_device_view const> col_desc,
                                       uint32_t num_rows,
                                       uint32_t fragment_size)
{
  gpu::InitPageFragments(frag, col_desc, fragment_size, num_rows, stream);
  frag.device_to_host(stream, true);
}

void writer::impl::gather_fragment_statistics(
  device_2dspan<statistics_chunk> frag_stats_chunk,
  device_2dspan<gpu::PageFragment const> frag,
  device_span<gpu::parquet_column_device_view const> col_desc,
  uint32_t num_fragments)
{
  auto num_columns = col_desc.size();
  rmm::device_uvector<statistics_group> frag_stats_group(num_fragments * num_columns, stream);
  auto frag_stats_group_2dview =
    device_2dspan<statistics_group>(frag_stats_group.data(), num_columns, num_fragments);

  gpu::InitFragmentStatistics(frag_stats_group_2dview, frag, col_desc, stream);
  detail::calculate_group_statistics<detail::io_file_format::PARQUET>(
    frag_stats_chunk.data(), frag_stats_group.data(), num_fragments * num_columns, stream);
  stream.synchronize();
}

void writer::impl::build_chunk_dictionaries(
  hostdevice_2dvector<gpu::EncColumnChunk>& chunks,
  device_span<gpu::parquet_column_device_view const> col_desc,
  uint32_t num_columns,
  uint32_t num_dictionaries)
{
  chunks.host_to_device(stream);
  if (num_dictionaries > 0) {
    size_t dict_scratch_size = (size_t)num_dictionaries * gpu::kDictScratchSize;
    auto dict_scratch        = cudf::detail::make_zeroed_device_uvector_async<uint32_t>(
      dict_scratch_size / sizeof(uint32_t), stream);

    gpu::BuildChunkDictionaries(chunks.device_view().flat_view(), dict_scratch.data(), stream);
  }
  gpu::InitEncoderPages(chunks, {}, col_desc, num_columns, nullptr, nullptr, stream);
  chunks.device_to_host(stream, true);
}

void writer::impl::init_encoder_pages(hostdevice_2dvector<gpu::EncColumnChunk>& chunks,
                                      device_span<gpu::parquet_column_device_view const> col_desc,
                                      device_span<gpu::EncPage> pages,
                                      statistics_chunk* page_stats,
                                      statistics_chunk* frag_stats,
                                      uint32_t num_columns,
                                      uint32_t num_pages,
                                      uint32_t num_stats_bfr)
{
  rmm::device_uvector<statistics_merge_group> page_stats_mrg(num_stats_bfr, stream);
  chunks.host_to_device(stream);
  InitEncoderPages(chunks,
                   pages,
                   col_desc,
                   num_columns,
                   (num_stats_bfr) ? page_stats_mrg.data() : nullptr,
                   (num_stats_bfr > num_pages) ? page_stats_mrg.data() + num_pages : nullptr,
                   stream);
  if (num_stats_bfr > 0) {
    detail::merge_group_statistics<detail::io_file_format::PARQUET>(
      page_stats, frag_stats, page_stats_mrg.data(), num_pages, stream);
    if (num_stats_bfr > num_pages) {
      detail::merge_group_statistics<detail::io_file_format::PARQUET>(
        page_stats + num_pages,
        page_stats,
        page_stats_mrg.data() + num_pages,
        num_stats_bfr - num_pages,
        stream);
    }
  }
  stream.synchronize();
}

void writer::impl::encode_pages(hostdevice_2dvector<gpu::EncColumnChunk>& chunks,
                                device_span<gpu::EncPage> pages,
                                uint32_t pages_in_batch,
                                uint32_t first_page_in_batch,
                                uint32_t rowgroups_in_batch,
                                uint32_t first_rowgroup,
                                const statistics_chunk* page_stats,
                                const statistics_chunk* chunk_stats)
{
  auto batch_pages = pages.subspan(first_page_in_batch, pages_in_batch);

  auto batch_pages_stats =
    (page_stats != nullptr)
      ? device_span<statistics_chunk const>(page_stats + first_page_in_batch, pages_in_batch)
      : device_span<statistics_chunk const>();

  uint32_t max_comp_pages =
    (compression_ != parquet::Compression::UNCOMPRESSED) ? pages_in_batch : 0;

  rmm::device_uvector<gpu_inflate_input_s> compression_input(max_comp_pages, stream);
  rmm::device_uvector<gpu_inflate_status_s> compression_status(max_comp_pages, stream);

  device_span<gpu_inflate_input_s> comp_in{compression_input.data(), compression_input.size()};
  device_span<gpu_inflate_status_s> comp_stat{compression_status.data(), compression_status.size()};

  gpu::EncodePages(batch_pages, comp_in, comp_stat, stream);
  switch (compression_) {
    case parquet::Compression::SNAPPY:
      CUDA_TRY(gpu_snap(comp_in.data(), comp_stat.data(), pages_in_batch, stream));
      break;
    default: break;
  }
  // TBD: Not clear if the official spec actually allows dynamically turning off compression at the
  // chunk-level
  auto d_chunks_in_batch = chunks.device_view().subspan(first_rowgroup, rowgroups_in_batch);
  DecideCompression(d_chunks_in_batch.flat_view(), stream);
  EncodePageHeaders(batch_pages, comp_stat, batch_pages_stats, chunk_stats, stream);
  GatherPages(d_chunks_in_batch.flat_view(), pages, stream);

  auto h_chunks_in_batch = chunks.host_view().subspan(first_rowgroup, rowgroups_in_batch);
  CUDA_TRY(hipMemcpyAsync(h_chunks_in_batch.data(),
                           d_chunks_in_batch.data(),
                           d_chunks_in_batch.flat_view().size_bytes(),
                           hipMemcpyDeviceToHost,
                           stream.value()));
  stream.synchronize();
}

writer::impl::impl(std::unique_ptr<data_sink> sink,
                   parquet_writer_options const& options,
                   SingleWriteMode mode,
                   rmm::cuda_stream_view stream,
                   rmm::mr::device_memory_resource* mr)
  : _mr(mr),
    stream(stream),
    compression_(to_parquet_compression(options.get_compression())),
    stats_granularity_(options.get_stats_level()),
    int96_timestamps(options.is_enabled_int96_timestamps()),
    out_sink_(std::move(sink)),
    single_write_mode(mode == SingleWriteMode::YES)
{
  if (options.get_metadata()) {
    table_meta = std::make_unique<table_input_metadata>(*options.get_metadata());
  }
  init_state();
}

writer::impl::impl(std::unique_ptr<data_sink> sink,
                   chunked_parquet_writer_options const& options,
                   SingleWriteMode mode,
                   rmm::cuda_stream_view stream,
                   rmm::mr::device_memory_resource* mr)
  : _mr(mr),
    stream(stream),
    compression_(to_parquet_compression(options.get_compression())),
    stats_granularity_(options.get_stats_level()),
    int96_timestamps(options.is_enabled_int96_timestamps()),
    single_write_mode(mode == SingleWriteMode::YES),
    out_sink_(std::move(sink))
{
  if (options.get_metadata()) {
    table_meta = std::make_unique<table_input_metadata>(*options.get_metadata());
  }
  init_state();
}

writer::impl::~impl() { close(); }

void writer::impl::init_state()
{
  // Write file header
  file_header_s fhdr;
  fhdr.magic = parquet_magic;
  out_sink_->host_write(&fhdr, sizeof(fhdr));
  current_chunk_offset = sizeof(file_header_s);
}

void writer::impl::write(table_view const& table)
{
  CUDF_EXPECTS(not closed, "Data has already been flushed to out and closed");

  size_type num_rows = table.num_rows();

  if (not table_meta) { table_meta = std::make_unique<table_input_metadata>(table); }

  // Fill unnamed columns' names in table_meta
  std::function<void(column_in_metadata&, std::string)> add_default_name =
    [&](column_in_metadata& col_meta, std::string default_name) {
      if (col_meta.get_name().empty()) col_meta.set_name(default_name);
      for (size_type i = 0; i < col_meta.num_children(); ++i) {
        add_default_name(col_meta.child(i), col_meta.get_name() + "_" + std::to_string(i));
      }
    };
  for (size_t i = 0; i < table_meta->column_metadata.size(); ++i) {
    add_default_name(table_meta->column_metadata[i], "_col" + std::to_string(i));
  }

  auto vec         = input_table_to_linked_columns(table);
  auto schema_tree = construct_schema_tree(vec, *table_meta, single_write_mode, int96_timestamps);
  // Construct parquet_column_views from the schema tree leaf nodes.
  std::vector<parquet_column_view> parquet_columns;

  for (schema_tree_node const& schema_node : schema_tree) {
    if (schema_node.leaf_column) { parquet_columns.emplace_back(schema_node, schema_tree, stream); }
  }

  // Mass allocation of column_device_views for each parquet_column_view
  std::vector<column_view> cudf_cols;
  cudf_cols.reserve(parquet_columns.size());
  for (auto const& parq_col : parquet_columns) {
    cudf_cols.push_back(parq_col.cudf_column_view());
  }
  table_view single_streams_table(cudf_cols);
  size_type num_columns = single_streams_table.num_columns();

  std::vector<SchemaElement> this_table_schema(schema_tree.begin(), schema_tree.end());

  if (md.version == 0) {
    md.version  = 1;
    md.num_rows = num_rows;
    md.column_order_listsize =
      (stats_granularity_ != statistics_freq::STATISTICS_NONE) ? num_columns : 0;
    std::transform(table_meta->user_data.begin(),
                   table_meta->user_data.end(),
                   std::back_inserter(md.key_value_metadata),
                   [](auto const& kv) {
                     return KeyValue{kv.first, kv.second};
                   });
    md.schema = this_table_schema;
  } else {
    // verify the user isn't passing mismatched tables
    CUDF_EXPECTS(md.schema == this_table_schema,
                 "Mismatch in schema between multiple calls to write_chunk");

    // increment num rows
    md.num_rows += num_rows;
  }
  // Create table_device_view so that corresponding column_device_view data
  // can be written into col_desc members
  auto parent_column_table_device_view = table_device_view::create(single_streams_table, stream);
  rmm::device_uvector<column_device_view> leaf_column_views(0, stream);

  // Initialize column description
  hostdevice_vector<gpu::parquet_column_device_view> col_desc(parquet_columns.size(), stream);
  // This should've been `auto const&` but isn't since dictionary space is allocated when calling
  // get_device_view(). Fix during dictionary refactor.
  std::transform(
    parquet_columns.begin(), parquet_columns.end(), col_desc.host_ptr(), [&](auto& pcol) {
      return pcol.get_device_view(stream);
    });

  // Init page fragments
  // 5000 is good enough for up to ~200-character strings. Longer strings will start producing
  // fragments larger than the desired page size -> TODO: keep track of the max fragment size, and
  // iteratively reduce this value if the largest fragment exceeds the max page size limit (we
  // ideally want the page size to be below 1MB so as to have enough pages to get good
  // compression/decompression performance).
  using cudf::io::parquet::gpu::max_page_fragment_size;
  constexpr uint32_t fragment_size = 5000;
  static_assert(fragment_size <= max_page_fragment_size,
                "fragment size cannot be greater than max_page_fragment_size");

  uint32_t num_fragments = (uint32_t)((num_rows + fragment_size - 1) / fragment_size);
  cudf::detail::hostdevice_2dvector<gpu::PageFragment> fragments(
    num_columns, num_fragments, stream);

  if (num_fragments != 0) {
    // Move column info to device
    col_desc.host_to_device(stream);
    leaf_column_views = create_leaf_column_device_views<gpu::parquet_column_device_view>(
      col_desc, *parent_column_table_device_view, stream);

    init_page_fragments(fragments, col_desc, num_rows, fragment_size);
  }

  size_t global_rowgroup_base = md.row_groups.size();

  // Decide row group boundaries based on uncompressed data size
  size_t rowgroup_size   = 0;
  uint32_t num_rowgroups = 0;
  for (uint32_t f = 0, global_r = global_rowgroup_base, rowgroup_start = 0; f < num_fragments;
       f++) {
    size_t fragment_data_size = 0;
    // Replace with STL algorithm to transform and sum
    for (auto i = 0; i < num_columns; i++) {
      fragment_data_size += fragments[i][f].fragment_data_size;
    }
    if (f > rowgroup_start && (rowgroup_size + fragment_data_size > max_rowgroup_size_ ||
                               (f + 1 - rowgroup_start) * fragment_size > max_rowgroup_rows_)) {
      // update schema
      md.row_groups.resize(md.row_groups.size() + 1);
      md.row_groups[global_r++].num_rows = (f - rowgroup_start) * fragment_size;
      num_rowgroups++;
      rowgroup_start = f;
      rowgroup_size  = 0;
    }
    rowgroup_size += fragment_data_size;
    if (f + 1 == num_fragments) {
      // update schema
      md.row_groups.resize(md.row_groups.size() + 1);
      md.row_groups[global_r++].num_rows = num_rows - rowgroup_start * fragment_size;
      num_rowgroups++;
    }
  }

  // Allocate column chunks and gather fragment statistics
  rmm::device_uvector<statistics_chunk> frag_stats(0, stream);
  if (stats_granularity_ != statistics_freq::STATISTICS_NONE) {
    frag_stats.resize(num_fragments * num_columns, stream);
    if (frag_stats.size() != 0) {
      auto frag_stats_2dview =
        device_2dspan<statistics_chunk>(frag_stats.data(), num_columns, num_fragments);
      gather_fragment_statistics(frag_stats_2dview, fragments, col_desc, num_fragments);
    }
  }
  // Initialize row groups and column chunks
  uint32_t num_chunks = num_rowgroups * num_columns;
  hostdevice_2dvector<gpu::EncColumnChunk> chunks(num_rowgroups, num_columns, stream);
  uint32_t num_dictionaries = 0;
  for (uint32_t r = 0, global_r = global_rowgroup_base, f = 0, start_row = 0; r < num_rowgroups;
       r++, global_r++) {
    uint32_t fragments_in_chunk =
      (uint32_t)((md.row_groups[global_r].num_rows + fragment_size - 1) / fragment_size);
    md.row_groups[global_r].total_byte_size = 0;
    md.row_groups[global_r].columns.resize(num_columns);
    for (int i = 0; i < num_columns; i++) {
      gpu::EncColumnChunk* ck = &chunks[r][i];
      bool dict_enable        = false;

      *ck           = {};
      ck->col_desc  = col_desc.device_ptr() + i;
      ck->fragments = &fragments.device_view()[i][f];
      ck->stats = (frag_stats.size() != 0) ? frag_stats.data() + i * num_fragments + f : nullptr;
      ck->start_row        = start_row;
      ck->num_rows         = (uint32_t)md.row_groups[global_r].num_rows;
      ck->first_fragment   = i * num_fragments + f;
      auto chunk_fragments = fragments[i].subspan(f, fragments_in_chunk);
      ck->num_values =
        std::accumulate(chunk_fragments.begin(), chunk_fragments.end(), 0, [](uint32_t l, auto r) {
          return l + r.num_values;
        });
      ck->dictionary_id = num_dictionaries;
      if (col_desc[i].dict_data) {
        size_t plain_size      = 0;
        size_t dict_size       = 1;
        uint32_t num_dict_vals = 0;
        for (uint32_t j = 0; j < fragments_in_chunk && num_dict_vals < 65536; j++) {
          plain_size += chunk_fragments[j].fragment_data_size;
          dict_size += chunk_fragments[j].dict_data_size +
                       ((num_dict_vals > 256) ? 2 : 1) * chunk_fragments[j].non_nulls;
          num_dict_vals += chunk_fragments[j].num_dict_vals;
        }
        if (dict_size < plain_size) {
          parquet_columns[i].use_dictionary(true);
          dict_enable = true;
          num_dictionaries++;
        }
      }
      ck->has_dictionary                                     = dict_enable;
      md.row_groups[global_r].columns[i].meta_data.type      = parquet_columns[i].physical_type();
      md.row_groups[global_r].columns[i].meta_data.encodings = {Encoding::PLAIN, Encoding::RLE};
      if (dict_enable) {
        md.row_groups[global_r].columns[i].meta_data.encodings.push_back(
          Encoding::PLAIN_DICTIONARY);
      }
      md.row_groups[global_r].columns[i].meta_data.path_in_schema =
        parquet_columns[i].get_path_in_schema();
      md.row_groups[global_r].columns[i].meta_data.codec      = UNCOMPRESSED;
      md.row_groups[global_r].columns[i].meta_data.num_values = ck->num_values;
    }
    f += fragments_in_chunk;
    start_row += (uint32_t)md.row_groups[global_r].num_rows;
  }

  // Free unused dictionaries
  for (auto& col : parquet_columns) {
    col.check_dictionary_used(stream);
  }

  // Build chunk dictionaries and count pages
  if (num_chunks != 0) {
    build_chunk_dictionaries(chunks, col_desc, num_columns, num_dictionaries);
  }

  // Initialize batches of rowgroups to encode (mainly to limit peak memory usage)
  std::vector<uint32_t> batch_list;
  uint32_t num_pages          = 0;
  size_t max_bytes_in_batch   = 1024 * 1024 * 1024;  // 1GB - TBD: Tune this
  size_t max_uncomp_bfr_size  = 0;
  size_t max_chunk_bfr_size   = 0;
  uint32_t max_pages_in_batch = 0;
  size_t bytes_in_batch       = 0;
  for (uint32_t r = 0, groups_in_batch = 0, pages_in_batch = 0; r <= num_rowgroups; r++) {
    size_t rowgroup_size = 0;
    if (r < num_rowgroups) {
      for (int i = 0; i < num_columns; i++) {
        gpu::EncColumnChunk* ck = &chunks[r][i];
        ck->first_page          = num_pages;
        num_pages += ck->num_pages;
        pages_in_batch += ck->num_pages;
        rowgroup_size += ck->bfr_size;
        max_chunk_bfr_size =
          std::max(max_chunk_bfr_size, (size_t)std::max(ck->bfr_size, ck->compressed_size));
      }
    }
    // TBD: We may want to also shorten the batch if we have enough pages (not just based on size)
    if ((r == num_rowgroups) ||
        (groups_in_batch != 0 && bytes_in_batch + rowgroup_size > max_bytes_in_batch)) {
      max_uncomp_bfr_size = std::max(max_uncomp_bfr_size, bytes_in_batch);
      max_pages_in_batch  = std::max(max_pages_in_batch, pages_in_batch);
      if (groups_in_batch != 0) {
        batch_list.push_back(groups_in_batch);
        groups_in_batch = 0;
      }
      bytes_in_batch = 0;
      pages_in_batch = 0;
    }
    bytes_in_batch += rowgroup_size;
    groups_in_batch++;
  }

  // Initialize data pointers in batch
  size_t max_comp_bfr_size =
    (compression_ != parquet::Compression::UNCOMPRESSED)
      ? gpu::GetMaxCompressedBfrSize(max_uncomp_bfr_size, max_pages_in_batch)
      : 0;
  uint32_t num_stats_bfr =
    (stats_granularity_ != statistics_freq::STATISTICS_NONE) ? num_pages + num_chunks : 0;
  rmm::device_buffer uncomp_bfr(max_uncomp_bfr_size, stream);
  rmm::device_buffer comp_bfr(max_comp_bfr_size, stream);
  rmm::device_uvector<gpu::EncPage> pages(num_pages, stream);

  // This contains stats for both the pages and the rowgroups. TODO: make them separate.
  rmm::device_uvector<statistics_chunk> page_stats(num_stats_bfr, stream);
  for (uint32_t b = 0, r = 0; b < (uint32_t)batch_list.size(); b++) {
    uint8_t* bfr   = static_cast<uint8_t*>(uncomp_bfr.data());
    uint8_t* bfr_c = static_cast<uint8_t*>(comp_bfr.data());
    for (uint32_t j = 0; j < batch_list[b]; j++, r++) {
      for (int i = 0; i < num_columns; i++) {
        gpu::EncColumnChunk* ck = &chunks[r][i];
        ck->uncompressed_bfr    = bfr;
        ck->compressed_bfr      = bfr_c;
        bfr += ck->bfr_size;
        bfr_c += ck->compressed_size;
      }
    }
  }

  if (num_pages != 0) {
    init_encoder_pages(chunks,
                       col_desc,
                       {pages.data(), pages.size()},
                       (num_stats_bfr) ? page_stats.data() : nullptr,
                       (num_stats_bfr) ? frag_stats.data() : nullptr,
                       num_columns,
                       num_pages,
                       num_stats_bfr);
  }

  pinned_buffer<uint8_t> host_bfr{nullptr, hipHostFree};

  // Encode row groups in batches
  for (uint32_t b = 0, r = 0, global_r = global_rowgroup_base; b < (uint32_t)batch_list.size();
       b++) {
    // Count pages in this batch
    uint32_t rnext               = r + batch_list[b];
    uint32_t first_page_in_batch = chunks[r][0].first_page;
    uint32_t first_page_in_next_batch =
      (rnext < num_rowgroups) ? chunks[rnext][0].first_page : num_pages;
    uint32_t pages_in_batch = first_page_in_next_batch - first_page_in_batch;
    // device_span<gpu::EncPage> batch_pages{pages.data() + first_page_in_batch, }
    encode_pages(
      chunks,
      {pages.data(), pages.size()},
      pages_in_batch,
      first_page_in_batch,
      batch_list[b],
      r,
      (stats_granularity_ == statistics_freq::STATISTICS_PAGE) ? page_stats.data() : nullptr,
      (stats_granularity_ != statistics_freq::STATISTICS_NONE) ? page_stats.data() + num_pages
                                                               : nullptr);
    for (; r < rnext; r++, global_r++) {
      for (auto i = 0; i < num_columns; i++) {
        gpu::EncColumnChunk* ck = &chunks[r][i];
        uint8_t* dev_bfr;
        if (ck->is_compressed) {
          md.row_groups[global_r].columns[i].meta_data.codec = compression_;
          dev_bfr                                            = ck->compressed_bfr;
        } else {
          dev_bfr = ck->uncompressed_bfr;
        }

        if (out_sink_->is_device_write_preferred(ck->compressed_size)) {
          // let the writer do what it wants to retrieve the data from the gpu.
          out_sink_->device_write(dev_bfr + ck->ck_stat_size, ck->compressed_size, stream);
          // we still need to do a (much smaller) memcpy for the statistics.
          if (ck->ck_stat_size != 0) {
            md.row_groups[global_r].columns[i].meta_data.statistics_blob.resize(ck->ck_stat_size);
            CUDA_TRY(
              hipMemcpyAsync(md.row_groups[global_r].columns[i].meta_data.statistics_blob.data(),
                              dev_bfr,
                              ck->ck_stat_size,
                              hipMemcpyDeviceToHost,
                              stream.value()));
            stream.synchronize();
          }
        } else {
          if (!host_bfr) {
            host_bfr = pinned_buffer<uint8_t>{[](size_t size) {
                                                uint8_t* ptr = nullptr;
                                                CUDA_TRY(hipHostMalloc(&ptr, size));
                                                return ptr;
                                              }(max_chunk_bfr_size),
                                              hipHostFree};
          }
          // copy the full data
          CUDA_TRY(hipMemcpyAsync(host_bfr.get(),
                                   dev_bfr,
                                   ck->ck_stat_size + ck->compressed_size,
                                   hipMemcpyDeviceToHost,
                                   stream.value()));
          stream.synchronize();
          out_sink_->host_write(host_bfr.get() + ck->ck_stat_size, ck->compressed_size);
          if (ck->ck_stat_size != 0) {
            md.row_groups[global_r].columns[i].meta_data.statistics_blob.resize(ck->ck_stat_size);
            memcpy(md.row_groups[global_r].columns[i].meta_data.statistics_blob.data(),
                   host_bfr.get(),
                   ck->ck_stat_size);
          }
        }
        md.row_groups[global_r].total_byte_size += ck->compressed_size;
        md.row_groups[global_r].columns[i].meta_data.data_page_offset =
          current_chunk_offset + ((ck->has_dictionary) ? ck->dictionary_size : 0);
        md.row_groups[global_r].columns[i].meta_data.dictionary_page_offset =
          (ck->has_dictionary) ? current_chunk_offset : 0;
        md.row_groups[global_r].columns[i].meta_data.total_uncompressed_size = ck->bfr_size;
        md.row_groups[global_r].columns[i].meta_data.total_compressed_size   = ck->compressed_size;
        current_chunk_offset += ck->compressed_size;
      }
    }
  }
}

std::unique_ptr<std::vector<uint8_t>> writer::impl::close(
  std::string const& column_chunks_file_path)
{
  if (closed) { return nullptr; }
  closed = true;
  CompactProtocolWriter cpw(&buffer_);
  file_ender_s fendr;
  buffer_.resize(0);
  fendr.footer_len = static_cast<uint32_t>(cpw.write(md));
  fendr.magic      = parquet_magic;
  out_sink_->host_write(buffer_.data(), buffer_.size());
  out_sink_->host_write(&fendr, sizeof(fendr));
  out_sink_->flush();

  // Optionally output raw file metadata with the specified column chunk file path
  if (column_chunks_file_path.length() > 0) {
    file_header_s fhdr = {parquet_magic};
    buffer_.resize(0);
    buffer_.insert(buffer_.end(),
                   reinterpret_cast<const uint8_t*>(&fhdr),
                   reinterpret_cast<const uint8_t*>(&fhdr) + sizeof(fhdr));
    for (auto& rowgroup : md.row_groups) {
      for (auto& col : rowgroup.columns) {
        col.file_path = column_chunks_file_path;
      }
    }
    fendr.footer_len = static_cast<uint32_t>(cpw.write(md));
    buffer_.insert(buffer_.end(),
                   reinterpret_cast<const uint8_t*>(&fendr),
                   reinterpret_cast<const uint8_t*>(&fendr) + sizeof(fendr));
    return std::make_unique<std::vector<uint8_t>>(std::move(buffer_));
  } else {
    return {nullptr};
  }
}

// Forward to implementation
writer::writer(std::unique_ptr<data_sink> sink,
               parquet_writer_options const& options,
               SingleWriteMode mode,
               rmm::cuda_stream_view stream,
               rmm::mr::device_memory_resource* mr)
  : _impl(std::make_unique<impl>(std::move(sink), options, mode, stream, mr))
{
}

writer::writer(std::unique_ptr<data_sink> sink,
               chunked_parquet_writer_options const& options,
               SingleWriteMode mode,
               rmm::cuda_stream_view stream,
               rmm::mr::device_memory_resource* mr)
  : _impl(std::make_unique<impl>(std::move(sink), options, mode, stream, mr))
{
}

// Destructor within this translation unit
writer::~writer() = default;

// Forward to implementation
void writer::write(table_view const& table) { _impl->write(table); }

// Forward to implementation
std::unique_ptr<std::vector<uint8_t>> writer::close(std::string const& column_chunks_file_path)
{
  return _impl->close(column_chunks_file_path);
}

std::unique_ptr<std::vector<uint8_t>> writer::merge_rowgroup_metadata(
  const std::vector<std::unique_ptr<std::vector<uint8_t>>>& metadata_list)
{
  std::vector<uint8_t> output;
  CompactProtocolWriter cpw(&output);
  FileMetaData md;

  md.row_groups.reserve(metadata_list.size());
  for (const auto& blob : metadata_list) {
    CompactProtocolReader cpreader(
      blob.get()->data(),
      std::max<size_t>(blob.get()->size(), sizeof(file_ender_s)) - sizeof(file_ender_s));
    cpreader.skip_bytes(sizeof(file_header_s));  // Skip over file header
    if (md.num_rows == 0) {
      cpreader.read(&md);
    } else {
      FileMetaData tmp;
      cpreader.read(&tmp);
      md.row_groups.insert(md.row_groups.end(),
                           std::make_move_iterator(tmp.row_groups.begin()),
                           std::make_move_iterator(tmp.row_groups.end()));
      md.num_rows += tmp.num_rows;
    }
  }
  // Reader doesn't currently populate column_order, so infer it here
  if (md.row_groups.size() != 0) {
    uint32_t num_columns = static_cast<uint32_t>(md.row_groups[0].columns.size());
    md.column_order_listsize =
      (num_columns > 0 && md.row_groups[0].columns[0].meta_data.statistics_blob.size())
        ? num_columns
        : 0;
  }
  // Thrift-encode the resulting output
  file_header_s fhdr;
  file_ender_s fendr;
  fhdr.magic = parquet_magic;
  output.insert(output.end(),
                reinterpret_cast<const uint8_t*>(&fhdr),
                reinterpret_cast<const uint8_t*>(&fhdr) + sizeof(fhdr));
  fendr.footer_len = static_cast<uint32_t>(cpw.write(md));
  fendr.magic      = parquet_magic;
  output.insert(output.end(),
                reinterpret_cast<const uint8_t*>(&fendr),
                reinterpret_cast<const uint8_t*>(&fendr) + sizeof(fendr));
  return std::make_unique<std::vector<uint8_t>>(std::move(output));
}

}  // namespace parquet
}  // namespace detail
}  // namespace io
}  // namespace cudf
