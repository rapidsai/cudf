#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019-2022, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

/**
 * @file writer_impl.cu
 * @brief cuDF-IO parquet writer class implementation
 */

#include "compact_protocol_reader.hpp"
#include "compact_protocol_writer.hpp"
#include "parquet_common.hpp"
#include "parquet_gpu.cuh"
#include "writer_impl.hpp"

#include <io/comp/nvcomp_adapter.hpp>
#include <io/statistics/column_statistics.cuh>
#include <io/utilities/column_utils.cuh>
#include <io/utilities/config_utils.hpp>

#include <cudf/column/column_device_view.cuh>
#include <cudf/detail/iterator.cuh>
#include <cudf/detail/utilities/linked_column.hpp>
#include <cudf/detail/utilities/vector_factories.hpp>
#include <cudf/lists/detail/dremel.hpp>
#include <cudf/lists/lists_column_view.hpp>
#include <cudf/null_mask.hpp>
#include <cudf/strings/strings_column_view.hpp>
#include <cudf/table/table_device_view.cuh>

#include <rmm/cuda_stream_view.hpp>
#include <rmm/device_buffer.hpp>
#include <rmm/device_scalar.hpp>
#include <rmm/device_uvector.hpp>

#include <thrust/binary_search.h>
#include <thrust/for_each.h>
#include <thrust/host_vector.h>
#include <thrust/iterator/zip_iterator.h>
#include <thrust/transform.h>
#include <thrust/tuple.h>

#include <algorithm>
#include <cstring>
#include <numeric>
#include <utility>

namespace cudf {
namespace io {
namespace detail {
namespace parquet {
using namespace cudf::io::parquet;
using namespace cudf::io;

namespace {
/**
 * @brief Helper for pinned host memory
 */
template <typename T>
using pinned_buffer = std::unique_ptr<T, decltype(&hipHostFree)>;

/**
 * @brief Function that translates GDF compression to parquet compression
 */
parquet::Compression to_parquet_compression(compression_type compression)
{
  switch (compression) {
    case compression_type::AUTO:
    case compression_type::SNAPPY: return parquet::Compression::SNAPPY;
    case compression_type::ZSTD: return parquet::Compression::ZSTD;
    case compression_type::NONE: return parquet::Compression::UNCOMPRESSED;
    default: CUDF_FAIL("Unsupported compression type");
  }
}

}  // namespace

struct aggregate_writer_metadata {
  aggregate_writer_metadata(std::vector<partition_info> const& partitions,
                            size_type num_columns,
                            std::vector<SchemaElement> schema,
                            statistics_freq stats_granularity,
                            std::vector<std::map<std::string, std::string>> const& kv_md)
    : version(1), schema(std::move(schema)), files(partitions.size())
  {
    for (size_t i = 0; i < partitions.size(); ++i) {
      this->files[i].num_rows = partitions[i].num_rows;
    }
    this->column_order_listsize =
      (stats_granularity != statistics_freq::STATISTICS_NONE) ? num_columns : 0;

    for (size_t p = 0; p < kv_md.size(); ++p) {
      std::transform(kv_md[p].begin(),
                     kv_md[p].end(),
                     std::back_inserter(this->files[p].key_value_metadata),
                     [](auto const& kv) {
                       return KeyValue{kv.first, kv.second};
                     });
    }
  }

  void update_files(std::vector<partition_info> const& partitions)
  {
    CUDF_EXPECTS(partitions.size() == this->files.size(),
                 "New partitions must be same size as previously passed number of partitions");
    for (size_t i = 0; i < partitions.size(); ++i) {
      this->files[i].num_rows += partitions[i].num_rows;
    }
  }

  FileMetaData get_metadata(size_t part)
  {
    CUDF_EXPECTS(part < files.size(), "Invalid part index queried");
    FileMetaData meta{};
    meta.version               = this->version;
    meta.schema                = this->schema;
    meta.num_rows              = this->files[part].num_rows;
    meta.row_groups            = this->files[part].row_groups;
    meta.key_value_metadata    = this->files[part].key_value_metadata;
    meta.created_by            = this->created_by;
    meta.column_order_listsize = this->column_order_listsize;
    return meta;
  }

  void set_file_paths(std::vector<std::string> const& column_chunks_file_path)
  {
    for (size_t p = 0; p < this->files.size(); ++p) {
      auto& file            = this->files[p];
      auto const& file_path = column_chunks_file_path[p];
      for (auto& rowgroup : file.row_groups) {
        for (auto& col : rowgroup.columns) {
          col.file_path = file_path;
        }
      }
    }
  }

  FileMetaData get_merged_metadata()
  {
    FileMetaData merged_md;
    for (size_t p = 0; p < this->files.size(); ++p) {
      auto& file = this->files[p];
      if (p == 0) {
        merged_md = this->get_metadata(0);
      } else {
        merged_md.row_groups.insert(merged_md.row_groups.end(),
                                    std::make_move_iterator(file.row_groups.begin()),
                                    std::make_move_iterator(file.row_groups.end()));
        merged_md.num_rows += file.num_rows;
      }
    }
    return merged_md;
  }

  std::vector<size_t> num_row_groups_per_file()
  {
    std::vector<size_t> global_rowgroup_base;
    std::transform(this->files.begin(),
                   this->files.end(),
                   std::back_inserter(global_rowgroup_base),
                   [](auto const& part) { return part.row_groups.size(); });
    return global_rowgroup_base;
  }

  [[nodiscard]] bool schema_matches(std::vector<SchemaElement> const& schema) const
  {
    return this->schema == schema;
  }
  auto& file(size_t p) { return files[p]; }
  [[nodiscard]] size_t num_files() const { return files.size(); }

 private:
  int32_t version = 0;
  std::vector<SchemaElement> schema;
  struct per_file_metadata {
    int64_t num_rows = 0;
    std::vector<RowGroup> row_groups;
    std::vector<KeyValue> key_value_metadata;
    std::vector<OffsetIndex> offset_indexes;
    std::vector<std::vector<uint8_t>> column_indexes;
  };
  std::vector<per_file_metadata> files;
  std::string created_by         = "";
  uint32_t column_order_listsize = 0;
};

/**
 * @brief Extends SchemaElement to add members required in constructing parquet_column_view
 *
 * Added members are:
 * 1. leaf_column: Pointer to leaf linked_column_view which points to the corresponding data stream
 *    of a leaf schema node. For non-leaf struct node, this is nullptr.
 * 2. stats_dtype: datatype for statistics calculation required for the data stream of a leaf node.
 * 3. ts_scale: scale to multiply or divide timestamp by in order to convert timestamp to parquet
 *    supported types
 */
struct schema_tree_node : public SchemaElement {
  cudf::detail::LinkedColPtr leaf_column;
  statistics_dtype stats_dtype;
  int32_t ts_scale;

  // TODO(fut): Think about making schema a class that holds a vector of schema_tree_nodes. The
  // function construct_schema_tree could be its constructor. It can have method to get the per
  // column nullability given a schema node index corresponding to a leaf schema. Much easier than
  // that is a method to get path in schema, given a leaf node
};

struct leaf_schema_fn {
  schema_tree_node& col_schema;
  cudf::detail::LinkedColPtr const& col;
  column_in_metadata const& col_meta;
  bool timestamp_is_int96;

  template <typename T>
  std::enable_if_t<std::is_same_v<T, bool>, void> operator()()
  {
    col_schema.type        = Type::BOOLEAN;
    col_schema.stats_dtype = statistics_dtype::dtype_bool;
  }

  template <typename T>
  std::enable_if_t<std::is_same_v<T, int8_t>, void> operator()()
  {
    col_schema.type           = Type::INT32;
    col_schema.converted_type = ConvertedType::INT_8;
    col_schema.stats_dtype    = statistics_dtype::dtype_int8;
  }

  template <typename T>
  std::enable_if_t<std::is_same_v<T, int16_t>, void> operator()()
  {
    col_schema.type           = Type::INT32;
    col_schema.converted_type = ConvertedType::INT_16;
    col_schema.stats_dtype    = statistics_dtype::dtype_int16;
  }

  template <typename T>
  std::enable_if_t<std::is_same_v<T, int32_t>, void> operator()()
  {
    col_schema.type        = Type::INT32;
    col_schema.stats_dtype = statistics_dtype::dtype_int32;
  }

  template <typename T>
  std::enable_if_t<std::is_same_v<T, int64_t>, void> operator()()
  {
    col_schema.type        = Type::INT64;
    col_schema.stats_dtype = statistics_dtype::dtype_int64;
  }

  template <typename T>
  std::enable_if_t<std::is_same_v<T, uint8_t>, void> operator()()
  {
    col_schema.type           = Type::INT32;
    col_schema.converted_type = ConvertedType::UINT_8;
    col_schema.stats_dtype    = statistics_dtype::dtype_int8;
  }

  template <typename T>
  std::enable_if_t<std::is_same_v<T, uint16_t>, void> operator()()
  {
    col_schema.type           = Type::INT32;
    col_schema.converted_type = ConvertedType::UINT_16;
    col_schema.stats_dtype    = statistics_dtype::dtype_int16;
  }

  template <typename T>
  std::enable_if_t<std::is_same_v<T, uint32_t>, void> operator()()
  {
    col_schema.type           = Type::INT32;
    col_schema.converted_type = ConvertedType::UINT_32;
    col_schema.stats_dtype    = statistics_dtype::dtype_int32;
  }

  template <typename T>
  std::enable_if_t<std::is_same_v<T, uint64_t>, void> operator()()
  {
    col_schema.type           = Type::INT64;
    col_schema.converted_type = ConvertedType::UINT_64;
    col_schema.stats_dtype    = statistics_dtype::dtype_int64;
  }

  template <typename T>
  std::enable_if_t<std::is_same_v<T, float>, void> operator()()
  {
    col_schema.type        = Type::FLOAT;
    col_schema.stats_dtype = statistics_dtype::dtype_float32;
  }

  template <typename T>
  std::enable_if_t<std::is_same_v<T, double>, void> operator()()
  {
    col_schema.type        = Type::DOUBLE;
    col_schema.stats_dtype = statistics_dtype::dtype_float64;
  }

  template <typename T>
  std::enable_if_t<std::is_same_v<T, cudf::string_view>, void> operator()()
  {
    col_schema.type = Type::BYTE_ARRAY;
    if (col_meta.is_enabled_output_as_binary()) {
      col_schema.converted_type = ConvertedType::UNKNOWN;
      col_schema.stats_dtype    = statistics_dtype::dtype_byte_array;
    } else {
      col_schema.converted_type = ConvertedType::UTF8;
      col_schema.stats_dtype    = statistics_dtype::dtype_string;
    }
  }

  template <typename T>
  std::enable_if_t<std::is_same_v<T, cudf::timestamp_D>, void> operator()()
  {
    col_schema.type           = Type::INT32;
    col_schema.converted_type = ConvertedType::DATE;
    col_schema.stats_dtype    = statistics_dtype::dtype_int32;
  }

  template <typename T>
  std::enable_if_t<std::is_same_v<T, cudf::timestamp_s>, void> operator()()
  {
    col_schema.type = (timestamp_is_int96) ? Type::INT96 : Type::INT64;
    col_schema.converted_type =
      (timestamp_is_int96) ? ConvertedType::UNKNOWN : ConvertedType::TIMESTAMP_MILLIS;
    col_schema.stats_dtype = statistics_dtype::dtype_timestamp64;
    col_schema.ts_scale    = 1000;
  }

  template <typename T>
  std::enable_if_t<std::is_same_v<T, cudf::timestamp_ms>, void> operator()()
  {
    col_schema.type = (timestamp_is_int96) ? Type::INT96 : Type::INT64;
    col_schema.converted_type =
      (timestamp_is_int96) ? ConvertedType::UNKNOWN : ConvertedType::TIMESTAMP_MILLIS;
    col_schema.stats_dtype = statistics_dtype::dtype_timestamp64;
  }

  template <typename T>
  std::enable_if_t<std::is_same_v<T, cudf::timestamp_us>, void> operator()()
  {
    col_schema.type = (timestamp_is_int96) ? Type::INT96 : Type::INT64;
    col_schema.converted_type =
      (timestamp_is_int96) ? ConvertedType::UNKNOWN : ConvertedType::TIMESTAMP_MICROS;
    col_schema.stats_dtype = statistics_dtype::dtype_timestamp64;
  }

  template <typename T>
  std::enable_if_t<std::is_same_v<T, cudf::timestamp_ns>, void> operator()()
  {
    col_schema.type           = (timestamp_is_int96) ? Type::INT96 : Type::INT64;
    col_schema.converted_type = ConvertedType::UNKNOWN;
    col_schema.stats_dtype    = statistics_dtype::dtype_timestamp64;
    if (timestamp_is_int96) {
      col_schema.ts_scale = -1000;  // negative value indicates division by absolute value
    }
    // set logical type if it's not int96
    else {
      col_schema.logical_type.isset.TIMESTAMP            = true;
      col_schema.logical_type.TIMESTAMP.unit.isset.NANOS = true;
    }
  }

  //  unsupported outside cudf for parquet 1.0.
  template <typename T>
  std::enable_if_t<std::is_same_v<T, cudf::duration_D>, void> operator()()
  {
    col_schema.type                                = Type::INT32;
    col_schema.converted_type                      = ConvertedType::TIME_MILLIS;
    col_schema.stats_dtype                         = statistics_dtype::dtype_int32;
    col_schema.ts_scale                            = 24 * 60 * 60 * 1000;
    col_schema.logical_type.isset.TIME             = true;
    col_schema.logical_type.TIME.unit.isset.MILLIS = true;
  }

  template <typename T>
  std::enable_if_t<std::is_same_v<T, cudf::duration_s>, void> operator()()
  {
    col_schema.type                                = Type::INT32;
    col_schema.converted_type                      = ConvertedType::TIME_MILLIS;
    col_schema.stats_dtype                         = statistics_dtype::dtype_int32;
    col_schema.ts_scale                            = 1000;
    col_schema.logical_type.isset.TIME             = true;
    col_schema.logical_type.TIME.unit.isset.MILLIS = true;
  }

  template <typename T>
  std::enable_if_t<std::is_same_v<T, cudf::duration_ms>, void> operator()()
  {
    col_schema.type                                = Type::INT32;
    col_schema.converted_type                      = ConvertedType::TIME_MILLIS;
    col_schema.stats_dtype                         = statistics_dtype::dtype_int32;
    col_schema.logical_type.isset.TIME             = true;
    col_schema.logical_type.TIME.unit.isset.MILLIS = true;
  }

  template <typename T>
  std::enable_if_t<std::is_same_v<T, cudf::duration_us>, void> operator()()
  {
    col_schema.type                                = Type::INT64;
    col_schema.converted_type                      = ConvertedType::TIME_MICROS;
    col_schema.stats_dtype                         = statistics_dtype::dtype_int64;
    col_schema.logical_type.isset.TIME             = true;
    col_schema.logical_type.TIME.unit.isset.MICROS = true;
  }

  //  unsupported outside cudf for parquet 1.0.
  template <typename T>
  std::enable_if_t<std::is_same_v<T, cudf::duration_ns>, void> operator()()
  {
    col_schema.type                               = Type::INT64;
    col_schema.stats_dtype                        = statistics_dtype::dtype_int64;
    col_schema.logical_type.isset.TIME            = true;
    col_schema.logical_type.TIME.unit.isset.NANOS = true;
  }

  template <typename T>
  std::enable_if_t<cudf::is_fixed_point<T>(), void> operator()()
  {
    if (std::is_same_v<T, numeric::decimal32>) {
      col_schema.type              = Type::INT32;
      col_schema.stats_dtype       = statistics_dtype::dtype_int32;
      col_schema.decimal_precision = MAX_DECIMAL32_PRECISION;
    } else if (std::is_same_v<T, numeric::decimal64>) {
      col_schema.type              = Type::INT64;
      col_schema.stats_dtype       = statistics_dtype::dtype_decimal64;
      col_schema.decimal_precision = MAX_DECIMAL64_PRECISION;
    } else if (std::is_same_v<T, numeric::decimal128>) {
      col_schema.type              = Type::FIXED_LEN_BYTE_ARRAY;
      col_schema.type_length       = sizeof(__int128_t);
      col_schema.stats_dtype       = statistics_dtype::dtype_decimal128;
      col_schema.decimal_precision = MAX_DECIMAL128_PRECISION;
    } else {
      CUDF_FAIL("Unsupported fixed point type for parquet writer");
    }
    col_schema.converted_type = ConvertedType::DECIMAL;
    col_schema.decimal_scale = -col->type().scale();  // parquet and cudf disagree about scale signs
    if (col_meta.is_decimal_precision_set()) {
      CUDF_EXPECTS(col_meta.get_decimal_precision() >= col_schema.decimal_scale,
                   "Precision must be equal to or greater than scale!");
      col_schema.decimal_precision = col_meta.get_decimal_precision();
    }
  }

  template <typename T>
  std::enable_if_t<cudf::is_nested<T>(), void> operator()()
  {
    CUDF_FAIL("This functor is only meant for physical data types");
  }

  template <typename T>
  std::enable_if_t<cudf::is_dictionary<T>(), void> operator()()
  {
    CUDF_FAIL("Dictionary columns are not supported for writing");
  }
};

inline bool is_col_nullable(cudf::detail::LinkedColPtr const& col,
                            column_in_metadata const& col_meta,
                            bool single_write_mode)
{
  if (single_write_mode) {
    return col->nullable();
  } else {
    if (col_meta.is_nullability_defined()) {
      CUDF_EXPECTS(col_meta.nullable() || !col->nullable(),
                   "Mismatch in metadata prescribed nullability and input column nullability. "
                   "Metadata for nullable input column cannot prescribe nullability = false");
      return col_meta.nullable();
    } else {
      // For chunked write, when not provided nullability, we assume the worst case scenario
      // that all columns are nullable.
      return true;
    }
  }
}

/**
 * @brief Construct schema from input columns and per-column input options
 *
 * Recursively traverses through linked_columns and corresponding metadata to construct schema tree.
 * The resulting schema tree is stored in a vector in pre-order traversal order.
 */
std::vector<schema_tree_node> construct_schema_tree(
  cudf::detail::LinkedColVector const& linked_columns,
  table_input_metadata& metadata,
  bool single_write_mode,
  bool int96_timestamps)
{
  std::vector<schema_tree_node> schema;
  schema_tree_node root{};
  root.type            = UNDEFINED_TYPE;
  root.repetition_type = NO_REPETITION_TYPE;
  root.name            = "schema";
  root.num_children    = linked_columns.size();
  root.parent_idx      = -1;  // root schema has no parent
  schema.push_back(std::move(root));

  std::function<void(cudf::detail::LinkedColPtr const&, column_in_metadata&, size_t)> add_schema =
    [&](cudf::detail::LinkedColPtr const& col, column_in_metadata& col_meta, size_t parent_idx) {
      bool col_nullable = is_col_nullable(col, col_meta, single_write_mode);

      auto set_field_id = [&schema, parent_idx](schema_tree_node& s,
                                                column_in_metadata const& col_meta) {
        if (schema[parent_idx].name != "list" and col_meta.is_parquet_field_id_set()) {
          s.field_id = col_meta.get_parquet_field_id();
        }
      };

      auto is_last_list_child = [](cudf::detail::LinkedColPtr col) {
        if (col->type().id() != type_id::LIST) { return false; }
        auto const child_col_type =
          col->children[lists_column_view::child_column_index]->type().id();
        return child_col_type == type_id::UINT8;
      };

      // There is a special case for a list<int8> column with one byte column child. This column can
      // have a special flag that indicates we write this out as binary instead of a list. This is a
      // more efficient storage mechanism for a single-depth list of bytes, but is a departure from
      // original cuIO behavior so it is locked behind the option. If the option is selected on a
      // column that isn't a single-depth list<int8> the code will throw.
      if (col_meta.is_enabled_output_as_binary() && is_last_list_child(col)) {
        CUDF_EXPECTS(col_meta.num_children() == 2 or col_meta.num_children() == 0,
                     "Binary column's corresponding metadata should have zero or two children!");
        if (col_meta.num_children() > 0) {
          auto const data_col_type =
            col->children[lists_column_view::child_column_index]->type().id();

          CUDF_EXPECTS(col->children[lists_column_view::child_column_index]->children.size() == 0,
                       "Binary column must not be nested!");
        }

        schema_tree_node col_schema{};
        col_schema.type            = Type::BYTE_ARRAY;
        col_schema.converted_type  = ConvertedType::UNKNOWN;
        col_schema.stats_dtype     = statistics_dtype::dtype_byte_array;
        col_schema.repetition_type = col_nullable ? OPTIONAL : REQUIRED;
        col_schema.name = (schema[parent_idx].name == "list") ? "element" : col_meta.get_name();
        col_schema.parent_idx  = parent_idx;
        col_schema.leaf_column = col;
        set_field_id(col_schema, col_meta);
        col_schema.output_as_byte_array = col_meta.is_enabled_output_as_binary();
        schema.push_back(col_schema);
      } else if (col->type().id() == type_id::STRUCT) {
        // if struct, add current and recursively call for all children
        schema_tree_node struct_schema{};
        struct_schema.repetition_type =
          col_nullable ? FieldRepetitionType::OPTIONAL : FieldRepetitionType::REQUIRED;

        struct_schema.name = (schema[parent_idx].name == "list") ? "element" : col_meta.get_name();
        struct_schema.num_children = col->children.size();
        struct_schema.parent_idx   = parent_idx;
        set_field_id(struct_schema, col_meta);
        schema.push_back(std::move(struct_schema));

        auto struct_node_index = schema.size() - 1;
        // for (auto child_it = col->children.begin(); child_it < col->children.end(); child_it++) {
        //   add_schema(*child_it, struct_node_index);
        // }
        CUDF_EXPECTS(col->children.size() == static_cast<size_t>(col_meta.num_children()),
                     "Mismatch in number of child columns between input table and metadata");
        for (size_t i = 0; i < col->children.size(); ++i) {
          add_schema(col->children[i], col_meta.child(i), struct_node_index);
        }
      } else if (col->type().id() == type_id::LIST && !col_meta.is_map()) {
        // List schema is denoted by two levels for each nesting level and one final level for leaf.
        // The top level is the same name as the column name.
        // So e.g. List<List<int>> is denoted in the schema by
        // "col_name" : { "list" : { "element" : { "list" : { "element" } } } }

        schema_tree_node list_schema_1{};
        list_schema_1.converted_type = ConvertedType::LIST;
        list_schema_1.repetition_type =
          col_nullable ? FieldRepetitionType::OPTIONAL : FieldRepetitionType::REQUIRED;
        list_schema_1.name = (schema[parent_idx].name == "list") ? "element" : col_meta.get_name();
        list_schema_1.num_children = 1;
        list_schema_1.parent_idx   = parent_idx;
        set_field_id(list_schema_1, col_meta);
        schema.push_back(std::move(list_schema_1));

        schema_tree_node list_schema_2{};
        list_schema_2.repetition_type = FieldRepetitionType::REPEATED;
        list_schema_2.name            = "list";
        list_schema_2.num_children    = 1;
        list_schema_2.parent_idx      = schema.size() - 1;  // Parent is list_schema_1, last added.
        schema.push_back(std::move(list_schema_2));

        CUDF_EXPECTS(col_meta.num_children() == 2,
                     "List column's metadata should have exactly two children");

        add_schema(col->children[lists_column_view::child_column_index],
                   col_meta.child(lists_column_view::child_column_index),
                   schema.size() - 1);
      } else if (col->type().id() == type_id::LIST && col_meta.is_map()) {
        // Map schema is denoted by a list of struct
        // e.g. List<Struct<String,String>> will be
        // "col_name" : { "key_value" : { "key", "value" } }

        // verify the List child structure is a struct<left_child, right_child>
        column_view struct_col = *col->children[lists_column_view::child_column_index];
        CUDF_EXPECTS(struct_col.type().id() == type_id::STRUCT, "Map should be a List of struct");
        CUDF_EXPECTS(struct_col.num_children() == 2,
                     "Map should be a List of struct with two children only but found " +
                       std::to_string(struct_col.num_children()));

        schema_tree_node map_schema{};
        map_schema.converted_type = ConvertedType::MAP;
        map_schema.repetition_type =
          col_nullable ? FieldRepetitionType::OPTIONAL : FieldRepetitionType::REQUIRED;
        map_schema.name = col_meta.get_name();
        if (col_meta.is_parquet_field_id_set()) {
          map_schema.field_id = col_meta.get_parquet_field_id();
        }
        map_schema.num_children = 1;
        map_schema.parent_idx   = parent_idx;
        schema.push_back(std::move(map_schema));

        schema_tree_node repeat_group{};
        repeat_group.repetition_type = FieldRepetitionType::REPEATED;
        repeat_group.name            = "key_value";
        repeat_group.num_children    = 2;
        repeat_group.parent_idx      = schema.size() - 1;  // Parent is map_schema, last added.
        schema.push_back(std::move(repeat_group));

        CUDF_EXPECTS(col_meta.num_children() == 2,
                     "List column's metadata should have exactly two children");
        CUDF_EXPECTS(col_meta.child(lists_column_view::child_column_index).num_children() == 2,
                     "Map struct column should have exactly two children");
        // verify the col meta of children of the struct have name key and value
        auto& left_child_meta = col_meta.child(lists_column_view::child_column_index).child(0);
        left_child_meta.set_name("key");
        left_child_meta.set_nullability(false);

        auto& right_child_meta = col_meta.child(lists_column_view::child_column_index).child(1);
        right_child_meta.set_name("value");
        // check the repetition type of key is required i.e. the col should be non-nullable
        auto key_col = col->children[lists_column_view::child_column_index]->children[0];
        CUDF_EXPECTS(!is_col_nullable(key_col, left_child_meta, single_write_mode),
                     "key column cannot be nullable. For chunked writing, explicitly set the "
                     "nullability to false in metadata");
        // process key
        size_type struct_col_index = schema.size() - 1;
        add_schema(key_col, left_child_meta, struct_col_index);
        // process value
        add_schema(col->children[lists_column_view::child_column_index]->children[1],
                   right_child_meta,
                   struct_col_index);

      } else {
        // if leaf, add current
        if (col->type().id() == type_id::STRING) {
          CUDF_EXPECTS(col_meta.num_children() == 2 or col_meta.num_children() == 0,
                       "String column's corresponding metadata should have zero or two children");
        } else {
          CUDF_EXPECTS(col_meta.num_children() == 0,
                       "Leaf column's corresponding metadata cannot have children");
        }

        schema_tree_node col_schema{};

        bool timestamp_is_int96 = int96_timestamps or col_meta.is_enabled_int96_timestamps();

        cudf::type_dispatcher(col->type(),
                              leaf_schema_fn{col_schema, col, col_meta, timestamp_is_int96});

        col_schema.repetition_type = col_nullable ? OPTIONAL : REQUIRED;
        col_schema.name = (schema[parent_idx].name == "list") ? "element" : col_meta.get_name();
        col_schema.parent_idx  = parent_idx;
        col_schema.leaf_column = col;
        set_field_id(col_schema, col_meta);
        schema.push_back(col_schema);
      }
    };

  CUDF_EXPECTS(metadata.column_metadata.size() == linked_columns.size(),
               "Mismatch in the number of columns and the corresponding metadata elements");
  // Add all linked_columns to schema using parent_idx = 0 (root)
  for (size_t i = 0; i < linked_columns.size(); ++i) {
    add_schema(linked_columns[i], metadata.column_metadata[i], 0);
  }

  return schema;
}

/**
 * @brief Class to store parquet specific information for one data stream.
 *
 * Contains information about a single data stream. In case of struct columns, a data stream is one
 * of the child leaf columns that contains data.
 * e.g. A column Struct<int, List<float>> contains 2 data streams:
 * - Struct<int>
 * - Struct<List<float>>
 *
 */
struct parquet_column_view {
  parquet_column_view(schema_tree_node const& schema_node,
                      std::vector<schema_tree_node> const& schema_tree,
                      rmm::cuda_stream_view stream);

  [[nodiscard]] column_view leaf_column_view() const;
  [[nodiscard]] gpu::parquet_column_device_view get_device_view(rmm::cuda_stream_view stream) const;

  [[nodiscard]] column_view cudf_column_view() const { return cudf_col; }
  [[nodiscard]] parquet::Type physical_type() const { return schema_node.type; }
  [[nodiscard]] parquet::ConvertedType converted_type() const { return schema_node.converted_type; }

  std::vector<std::string> const& get_path_in_schema() { return path_in_schema; }

  // LIST related member functions
  [[nodiscard]] uint8_t max_def_level() const noexcept { return _max_def_level; }
  [[nodiscard]] uint8_t max_rep_level() const noexcept { return _max_rep_level; }
  [[nodiscard]] bool is_list() const noexcept { return _is_list; }

 private:
  // Schema related members
  schema_tree_node schema_node;
  std::vector<std::string> path_in_schema;
  uint8_t _max_def_level = 0;
  uint8_t _max_rep_level = 0;
  rmm::device_uvector<uint8_t> _d_nullability;

  column_view cudf_col;

  // List-related members
  bool _is_list;
  rmm::device_uvector<size_type>
    _dremel_offsets;  ///< For each row, the absolute offset into the repetition and definition
                      ///< level vectors. O(num rows)
  rmm::device_uvector<uint8_t> _rep_level;
  rmm::device_uvector<uint8_t> _def_level;
  std::vector<uint8_t> _nullability;
  size_type _data_count = 0;
};

parquet_column_view::parquet_column_view(schema_tree_node const& schema_node,
                                         std::vector<schema_tree_node> const& schema_tree,
                                         rmm::cuda_stream_view stream)
  : schema_node(schema_node),
    _d_nullability(0, stream),
    _dremel_offsets(0, stream),
    _rep_level(0, stream),
    _def_level(0, stream)
{
  // Construct single inheritance column_view from linked_column_view
  auto curr_col                           = schema_node.leaf_column.get();
  column_view single_inheritance_cudf_col = *curr_col;
  while (curr_col->parent) {
    auto const& parent = *curr_col->parent;

    // For list columns, we still need to retain the offset child column.
    auto children =
      (parent.type().id() == type_id::LIST)
        ? std::vector<column_view>{*parent.children[lists_column_view::offsets_column_index],
                                   single_inheritance_cudf_col}
        : std::vector<column_view>{single_inheritance_cudf_col};

    single_inheritance_cudf_col = column_view(parent.type(),
                                              parent.size(),
                                              parent.head(),
                                              parent.null_mask(),
                                              UNKNOWN_NULL_COUNT,
                                              parent.offset(),
                                              children);

    curr_col = curr_col->parent;
  }
  cudf_col = single_inheritance_cudf_col;

  // Construct path_in_schema by travelling up in the schema_tree
  std::vector<std::string> path;
  auto curr_schema_node = schema_node;
  do {
    path.push_back(curr_schema_node.name);
    if (curr_schema_node.parent_idx != -1) {
      curr_schema_node = schema_tree[curr_schema_node.parent_idx];
    }
  } while (curr_schema_node.parent_idx != -1);
  path_in_schema = std::vector<std::string>(path.crbegin(), path.crend());

  // Calculate max definition level by counting the number of levels that are optional (nullable)
  // and max repetition level by counting the number of REPEATED levels in this column's hierarchy
  uint16_t max_def_level = 0;
  uint16_t max_rep_level = 0;
  curr_schema_node       = schema_node;
  while (curr_schema_node.parent_idx != -1) {
    if (curr_schema_node.repetition_type == parquet::REPEATED or
        curr_schema_node.repetition_type == parquet::OPTIONAL) {
      ++max_def_level;
    }
    if (curr_schema_node.repetition_type == parquet::REPEATED) { ++max_rep_level; }
    curr_schema_node = schema_tree[curr_schema_node.parent_idx];
  }
  CUDF_EXPECTS(max_def_level < 256, "Definition levels above 255 are not supported");
  CUDF_EXPECTS(max_rep_level < 256, "Definition levels above 255 are not supported");

  _max_def_level = max_def_level;
  _max_rep_level = max_rep_level;

  // Construct nullability vector using repetition_type from schema.
  std::vector<uint8_t> r_nullability;
  curr_schema_node = schema_node;
  while (curr_schema_node.parent_idx != -1) {
    if (not curr_schema_node.is_stub()) {
      r_nullability.push_back(curr_schema_node.repetition_type == FieldRepetitionType::OPTIONAL);
    }
    curr_schema_node = schema_tree[curr_schema_node.parent_idx];
  }
  _nullability = std::vector<uint8_t>(r_nullability.crbegin(), r_nullability.crend());
  // TODO(cp): Explore doing this for all columns in a single go outside this ctor. Maybe using
  // hostdevice_vector. Currently this involves a hipMemcpyAsync for each column.
  _d_nullability = cudf::detail::make_device_uvector_async(_nullability, stream);

  _is_list = (_max_rep_level > 0);

  if (cudf_col.size() == 0) { return; }

  if (_is_list) {
    // Top level column's offsets are not applied to all children. Get the effective offset and
    // size of the leaf column
    // Calculate row offset into dremel data (repetition/definition values) and the respective
    // definition and repetition levels
    cudf::detail::dremel_data dremel =
      get_dremel_data(cudf_col, _nullability, schema_node.output_as_byte_array, stream);
    _dremel_offsets = std::move(dremel.dremel_offsets);
    _rep_level      = std::move(dremel.rep_level);
    _def_level      = std::move(dremel.def_level);
    _data_count     = dremel.leaf_data_size;  // Needed for knowing what size dictionary to allocate

    stream.synchronize();
  } else {
    // For non-list struct, the size of the root column is the same as the size of the leaf column
    _data_count = cudf_col.size();
  }
}

column_view parquet_column_view::leaf_column_view() const
{
  if (!schema_node.output_as_byte_array) {
    auto col = cudf_col;
    while (cudf::is_nested(col.type())) {
      if (col.type().id() == type_id::LIST) {
        col = col.child(lists_column_view::child_column_index);
      } else if (col.type().id() == type_id::STRUCT) {
        col = col.child(0);  // Stored cudf_col has only one child if struct
      }
    }
    return col;
  } else {
    // TODO: investigate why the leaf node is computed twice instead of using the schema leaf node
    // for everything
    return *schema_node.leaf_column;
  }
}

gpu::parquet_column_device_view parquet_column_view::get_device_view(
  rmm::cuda_stream_view stream) const
{
  column_view col  = leaf_column_view();
  auto desc        = gpu::parquet_column_device_view{};  // Zero out all fields
  desc.stats_dtype = schema_node.stats_dtype;
  desc.ts_scale    = schema_node.ts_scale;

  if (is_list()) {
    desc.level_offsets = _dremel_offsets.data();
    desc.rep_values    = _rep_level.data();
    desc.def_values    = _def_level.data();
  }
  desc.num_rows             = cudf_col.size();
  desc.physical_type        = physical_type();
  desc.converted_type       = converted_type();
  desc.output_as_byte_array = schema_node.output_as_byte_array;

  desc.level_bits = CompactProtocolReader::NumRequiredBits(max_rep_level()) << 4 |
                    CompactProtocolReader::NumRequiredBits(max_def_level());
  desc.nullability = _d_nullability.data();
  return desc;
}

void writer::impl::init_page_fragments(cudf::detail::hostdevice_2dvector<gpu::PageFragment>& frag,
                                       device_span<gpu::parquet_column_device_view const> col_desc,
                                       host_span<partition_info const> partitions,
                                       device_span<int const> part_frag_offset,
                                       uint32_t fragment_size)
{
  auto d_partitions = cudf::detail::make_device_uvector_async(partitions, stream);
  gpu::InitPageFragments(frag, col_desc, d_partitions, part_frag_offset, fragment_size, stream);
  frag.device_to_host(stream, true);
}

void writer::impl::gather_fragment_statistics(
  device_2dspan<statistics_chunk> frag_stats_chunk,
  device_2dspan<gpu::PageFragment const> frag,
  device_span<gpu::parquet_column_device_view const> col_desc,
  uint32_t num_fragments)
{
  auto num_columns = col_desc.size();
  rmm::device_uvector<statistics_group> frag_stats_group(num_fragments * num_columns, stream);
  auto frag_stats_group_2dview =
    device_2dspan<statistics_group>(frag_stats_group.data(), num_columns, num_fragments);

  gpu::InitFragmentStatistics(frag_stats_group_2dview, frag, col_desc, stream);
  detail::calculate_group_statistics<detail::io_file_format::PARQUET>(frag_stats_chunk.data(),
                                                                      frag_stats_group.data(),
                                                                      num_fragments * num_columns,
                                                                      stream,
                                                                      int96_timestamps);
  stream.synchronize();
}

auto to_nvcomp_compression_type(Compression codec)
{
  if (codec == Compression::SNAPPY) return nvcomp::compression_type::SNAPPY;
  if (codec == Compression::ZSTD) return nvcomp::compression_type::ZSTD;
  CUDF_FAIL("Unsupported compression type");
}

auto page_alignment(Compression codec)
{
  if (codec == Compression::UNCOMPRESSED or
      nvcomp::is_compression_disabled(to_nvcomp_compression_type(codec))) {
    return 1u;
  }

  return 1u << nvcomp::compress_input_alignment_bits(to_nvcomp_compression_type(codec));
}

size_t max_compression_output_size(Compression codec, uint32_t compression_blocksize)
{
  if (codec == Compression::UNCOMPRESSED) return 0;

  return compress_max_output_chunk_size(to_nvcomp_compression_type(codec), compression_blocksize);
}

auto init_page_sizes(hostdevice_2dvector<gpu::EncColumnChunk>& chunks,
                     device_span<gpu::parquet_column_device_view const> col_desc,
                     uint32_t num_columns,
                     size_t max_page_size_bytes,
                     size_type max_page_size_rows,
                     Compression compression_codec,
                     rmm::cuda_stream_view stream)
{
  if (chunks.is_empty()) { return hostdevice_vector<size_type>{}; }

  chunks.host_to_device(stream);
  // Calculate number of pages and store in respective chunks
  gpu::InitEncoderPages(chunks,
                        {},
                        {},
                        {},
                        col_desc,
                        num_columns,
                        max_page_size_bytes,
                        max_page_size_rows,
                        page_alignment(compression_codec),
                        nullptr,
                        nullptr,
                        stream);
  chunks.device_to_host(stream, true);

  int num_pages = 0;
  for (auto& chunk : chunks.host_view().flat_view()) {
    chunk.first_page = num_pages;
    num_pages += chunk.num_pages;
  }
  chunks.host_to_device(stream);

  // Now that we know the number of pages, allocate an array to hold per page size and get it
  // populated
  hostdevice_vector<size_type> page_sizes(num_pages, stream);
  gpu::InitEncoderPages(chunks,
                        {},
                        page_sizes,
                        {},
                        col_desc,
                        num_columns,
                        max_page_size_bytes,
                        max_page_size_rows,
                        page_alignment(compression_codec),
                        nullptr,
                        nullptr,
                        stream);
  page_sizes.device_to_host(stream, true);

  // Get per-page max compressed size
  hostdevice_vector<size_type> comp_page_sizes(num_pages, stream);
  std::transform(page_sizes.begin(),
                 page_sizes.end(),
                 comp_page_sizes.begin(),
                 [compression_codec](auto page_size) {
                   return max_compression_output_size(compression_codec, page_size);
                 });
  comp_page_sizes.host_to_device(stream);

  // Use per-page max compressed size to calculate chunk.compressed_size
  gpu::InitEncoderPages(chunks,
                        {},
                        {},
                        comp_page_sizes,
                        col_desc,
                        num_columns,
                        max_page_size_bytes,
                        max_page_size_rows,
                        page_alignment(compression_codec),
                        nullptr,
                        nullptr,
                        stream);
  chunks.device_to_host(stream, true);
  return comp_page_sizes;
}

auto build_chunk_dictionaries(hostdevice_2dvector<gpu::EncColumnChunk>& chunks,
                              host_span<gpu::parquet_column_device_view const> col_desc,
                              device_2dspan<gpu::PageFragment const> frags,
                              rmm::cuda_stream_view stream)
{
  // At this point, we know all chunks and their sizes. We want to allocate dictionaries for each
  // chunk that can have dictionary

  auto h_chunks = chunks.host_view().flat_view();

  std::vector<rmm::device_uvector<size_type>> dict_data;
  std::vector<rmm::device_uvector<size_type>> dict_index;

  if (h_chunks.size() == 0) { return std::pair(std::move(dict_data), std::move(dict_index)); }

  // Allocate slots for each chunk
  std::vector<rmm::device_uvector<gpu::slot_type>> hash_maps_storage;
  hash_maps_storage.reserve(h_chunks.size());
  for (auto& chunk : h_chunks) {
    if (col_desc[chunk.col_desc_id].physical_type == Type::BOOLEAN ||
        (col_desc[chunk.col_desc_id].output_as_byte_array &&
         col_desc[chunk.col_desc_id].physical_type == Type::BYTE_ARRAY)) {
      chunk.use_dictionary = false;
    } else {
      chunk.use_dictionary = true;
      // cuCollections suggests using a hash map of size N * (1/0.7) = num_values * 1.43
      // https://github.com/NVIDIA/cuCollections/blob/3a49fc71/include/cuco/static_map.cuh#L190-L193
      auto& inserted_map   = hash_maps_storage.emplace_back(chunk.num_values * 1.43, stream);
      chunk.dict_map_slots = inserted_map.data();
      chunk.dict_map_size  = inserted_map.size();
    }
  }

  chunks.host_to_device(stream);

  gpu::initialize_chunk_hash_maps(chunks.device_view().flat_view(), stream);
  gpu::populate_chunk_hash_maps(frags, stream);

  chunks.device_to_host(stream, true);

  // Make decision about which chunks have dictionary
  for (auto& ck : h_chunks) {
    if (not ck.use_dictionary) { continue; }
    std::tie(ck.use_dictionary, ck.dict_rle_bits) = [&]() -> std::pair<bool, uint8_t> {
      // calculate size of chunk if dictionary is used

      // If we have N unique values then the idx for the last value is N - 1 and nbits is the number
      // of bits required to encode indices into the dictionary
      auto max_dict_index = (ck.num_dict_entries > 0) ? ck.num_dict_entries - 1 : 0;
      auto nbits          = std::max(CompactProtocolReader::NumRequiredBits(max_dict_index), 1);

      // We don't use dictionary if the indices are > MAX_DICT_BITS bits because that's the maximum
      // bitpacking bitsize we efficiently support
      if (nbits > MAX_DICT_BITS) { return {false, 0}; }

      auto rle_byte_size = util::div_rounding_up_safe(ck.num_values * nbits, 8);
      auto dict_enc_size = ck.uniq_data_size + rle_byte_size;
      if (ck.plain_data_size <= dict_enc_size) { return {false, 0}; }

      return {true, nbits};
    }();
  }

  // TODO: (enh) Deallocate hash map storage for chunks that don't use dict and clear pointers.

  dict_data.reserve(h_chunks.size());
  dict_index.reserve(h_chunks.size());
  for (auto& chunk : h_chunks) {
    if (not chunk.use_dictionary) { continue; }

    size_t dict_data_size     = std::min(MAX_DICT_SIZE, chunk.dict_map_size);
    auto& inserted_dict_data  = dict_data.emplace_back(dict_data_size, stream);
    auto& inserted_dict_index = dict_index.emplace_back(chunk.num_values, stream);
    chunk.dict_data           = inserted_dict_data.data();
    chunk.dict_index          = inserted_dict_index.data();
  }
  chunks.host_to_device(stream);
  gpu::collect_map_entries(chunks.device_view().flat_view(), stream);
  gpu::get_dictionary_indices(frags, stream);

  return std::pair(std::move(dict_data), std::move(dict_index));
}

void writer::impl::init_encoder_pages(hostdevice_2dvector<gpu::EncColumnChunk>& chunks,
                                      device_span<gpu::parquet_column_device_view const> col_desc,
                                      device_span<gpu::EncPage> pages,
                                      hostdevice_vector<size_type>& comp_page_sizes,
                                      statistics_chunk* page_stats,
                                      statistics_chunk* frag_stats,
                                      uint32_t num_columns,
                                      uint32_t num_pages,
                                      uint32_t num_stats_bfr)
{
  rmm::device_uvector<statistics_merge_group> page_stats_mrg(num_stats_bfr, stream);
  chunks.host_to_device(stream);
  InitEncoderPages(chunks,
                   pages,
                   {},
                   comp_page_sizes,
                   col_desc,
                   num_columns,
                   max_page_size_bytes,
                   max_page_size_rows,
                   page_alignment(compression_),
                   (num_stats_bfr) ? page_stats_mrg.data() : nullptr,
                   (num_stats_bfr > num_pages) ? page_stats_mrg.data() + num_pages : nullptr,
                   stream);
  if (num_stats_bfr > 0) {
    detail::merge_group_statistics<detail::io_file_format::PARQUET>(
      page_stats, frag_stats, page_stats_mrg.data(), num_pages, stream);
    if (num_stats_bfr > num_pages) {
      detail::merge_group_statistics<detail::io_file_format::PARQUET>(
        page_stats + num_pages,
        page_stats,
        page_stats_mrg.data() + num_pages,
        num_stats_bfr - num_pages,
        stream);
    }
  }
  stream.synchronize();
}

void writer::impl::encode_pages(hostdevice_2dvector<gpu::EncColumnChunk>& chunks,
                                device_span<gpu::EncPage> pages,
                                size_t max_page_uncomp_data_size,
                                uint32_t pages_in_batch,
                                uint32_t first_page_in_batch,
                                uint32_t rowgroups_in_batch,
                                uint32_t first_rowgroup,
                                const statistics_chunk* page_stats,
                                const statistics_chunk* chunk_stats,
                                const statistics_chunk* column_stats)
{
  auto batch_pages = pages.subspan(first_page_in_batch, pages_in_batch);

  auto batch_pages_stats =
    (page_stats != nullptr)
      ? device_span<statistics_chunk const>(page_stats + first_page_in_batch, pages_in_batch)
      : device_span<statistics_chunk const>();

  uint32_t max_comp_pages =
    (compression_ != parquet::Compression::UNCOMPRESSED) ? pages_in_batch : 0;

  rmm::device_uvector<device_span<uint8_t const>> comp_in(max_comp_pages, stream);
  rmm::device_uvector<device_span<uint8_t>> comp_out(max_comp_pages, stream);
  rmm::device_uvector<compression_result> comp_res(max_comp_pages, stream);
  thrust::fill(rmm::exec_policy(stream),
               comp_res.begin(),
               comp_res.end(),
               compression_result{0, compression_status::FAILURE});

  gpu::EncodePages(batch_pages, comp_in, comp_out, comp_res, stream);
  switch (compression_) {
    case parquet::Compression::SNAPPY:
      if (nvcomp::is_compression_disabled(nvcomp::compression_type::SNAPPY)) {
        gpu_snap(comp_in, comp_out, comp_res, stream);
      } else {
        nvcomp::batched_compress(
          nvcomp::compression_type::SNAPPY, comp_in, comp_out, comp_res, stream);
      }
      break;
    case parquet::Compression::ZSTD: {
      if (auto const reason = nvcomp::is_compression_disabled(nvcomp::compression_type::ZSTD);
          reason) {
        CUDF_FAIL("Compression error: " + reason.value());
      }
      nvcomp::batched_compress(nvcomp::compression_type::ZSTD, comp_in, comp_out, comp_res, stream);

      break;
    }
    case parquet::Compression::UNCOMPRESSED: break;
    default: CUDF_FAIL("invalid compression type");
  }

  // TBD: Not clear if the official spec actually allows dynamically turning off compression at the
  // chunk-level
  auto d_chunks_in_batch = chunks.device_view().subspan(first_rowgroup, rowgroups_in_batch);
  DecideCompression(d_chunks_in_batch.flat_view(), stream);
  EncodePageHeaders(batch_pages, comp_res, batch_pages_stats, chunk_stats, stream);
  GatherPages(d_chunks_in_batch.flat_view(), pages, stream);

  if (column_stats != nullptr) {
    EncodeColumnIndexes(d_chunks_in_batch.flat_view(),
                        {column_stats, pages.size()},
                        column_index_truncate_length,
                        stream);
  }

  auto h_chunks_in_batch = chunks.host_view().subspan(first_rowgroup, rowgroups_in_batch);
  CUDF_CUDA_TRY(hipMemcpyAsync(h_chunks_in_batch.data(),
                                d_chunks_in_batch.data(),
                                d_chunks_in_batch.flat_view().size_bytes(),
                                hipMemcpyDeviceToHost,
                                stream.value()));
  stream.synchronize();
}

size_t writer::impl::column_index_buffer_size(gpu::EncColumnChunk* ck) const
{
  // encoding the column index for a given chunk requires:
  //   each list (4 of them) requires 6 bytes of overhead
  //     (1 byte field header, 1 byte type, 4 bytes length)
  //   1 byte overhead for boundary_order
  //   1 byte overhead for termination
  //   sizeof(char) for boundary_order
  //   sizeof(bool) * num_pages for null_pages
  //   (ck_max_stats_len + 4) * num_pages * 2 for min/max values
  //     (each binary requires 4 bytes length + ck_max_stats_len)
  //   sizeof(int64_t) * num_pages for null_counts
  //
  // so 26 bytes overhead + sizeof(char) +
  //    (sizeof(bool) + sizeof(int64_t) + 2 * (4 + ck_max_stats_len)) * num_pages
  //
  // we already have ck->ck_stat_size = 48 + 2 * ck_max_stats_len
  // all of the overhead and non-stats data can fit in under 48 bytes
  //
  // so we can simply use ck_stat_size * num_pages
  //
  // add on some extra padding at the end (plus extra 7 bytes of alignment padding)
  // for scratch space to do stats truncation.
  //
  // calculating this per-chunk because the sizes can be wildly different.
  constexpr size_t padding = 7;
  return ck->ck_stat_size * ck->num_pages + column_index_truncate_length + padding;
}

size_t max_page_bytes(Compression compression, size_t max_page_size_bytes)
{
  if (compression == parquet::Compression::UNCOMPRESSED) { return max_page_size_bytes; }

  auto const ncomp_type   = to_nvcomp_compression_type(compression);
  auto const nvcomp_limit = nvcomp::is_compression_disabled(ncomp_type)
                              ? std::nullopt
                              : nvcomp::compress_max_allowed_chunk_size(ncomp_type);

  auto max_size = std::min(nvcomp_limit.value_or(max_page_size_bytes), max_page_size_bytes);
  // page size must fit in a 32-bit signed integer
  return std::min<size_t>(max_size, std::numeric_limits<int32_t>::max());
}

writer::impl::impl(std::vector<std::unique_ptr<data_sink>> sinks,
                   parquet_writer_options const& options,
                   SingleWriteMode mode,
                   rmm::cuda_stream_view stream,
                   rmm::mr::device_memory_resource* mr)
  : _mr(mr),
    stream(stream),
    compression_(to_parquet_compression(options.get_compression())),
    max_row_group_size{options.get_row_group_size_bytes()},
    max_row_group_rows{options.get_row_group_size_rows()},
    max_page_size_bytes(max_page_bytes(compression_, options.get_max_page_size_bytes())),
    max_page_size_rows(options.get_max_page_size_rows()),
    stats_granularity_(options.get_stats_level()),
    int96_timestamps(options.is_enabled_int96_timestamps()),
    column_index_truncate_length(options.get_column_index_truncate_length()),
    kv_md(options.get_key_value_metadata()),
    single_write_mode(mode == SingleWriteMode::YES),
    out_sink_(std::move(sinks))
{
  if (options.get_metadata()) {
    table_meta = std::make_unique<table_input_metadata>(*options.get_metadata());
  }
  init_state();
}

writer::impl::impl(std::vector<std::unique_ptr<data_sink>> sinks,
                   chunked_parquet_writer_options const& options,
                   SingleWriteMode mode,
                   rmm::cuda_stream_view stream,
                   rmm::mr::device_memory_resource* mr)
  : _mr(mr),
    stream(stream),
    compression_(to_parquet_compression(options.get_compression())),
    max_row_group_size{options.get_row_group_size_bytes()},
    max_row_group_rows{options.get_row_group_size_rows()},
    max_page_size_bytes(max_page_bytes(compression_, options.get_max_page_size_bytes())),
    max_page_size_rows(options.get_max_page_size_rows()),
    stats_granularity_(options.get_stats_level()),
    int96_timestamps(options.is_enabled_int96_timestamps()),
    column_index_truncate_length(options.get_column_index_truncate_length()),
    kv_md(options.get_key_value_metadata()),
    single_write_mode(mode == SingleWriteMode::YES),
    out_sink_(std::move(sinks))
{
  if (options.get_metadata()) {
    table_meta = std::make_unique<table_input_metadata>(*options.get_metadata());
  }
  init_state();
}

writer::impl::~impl() { close(); }

void writer::impl::init_state()
{
  current_chunk_offset.resize(out_sink_.size());
  // Write file header
  file_header_s fhdr;
  fhdr.magic = parquet_magic;
  for (auto& sink : out_sink_) {
    sink->host_write(&fhdr, sizeof(fhdr));
  }
  std::fill_n(current_chunk_offset.begin(), current_chunk_offset.size(), sizeof(file_header_s));
}

void writer::impl::write(table_view const& table, std::vector<partition_info> const& partitions)
{
  last_write_successful = false;
  CUDF_EXPECTS(not closed, "Data has already been flushed to out and closed");

  if (not table_meta) { table_meta = std::make_unique<table_input_metadata>(table); }

  // Fill unnamed columns' names in table_meta
  std::function<void(column_in_metadata&, std::string)> add_default_name =
    [&](column_in_metadata& col_meta, std::string default_name) {
      if (col_meta.get_name().empty()) col_meta.set_name(default_name);
      for (size_type i = 0; i < col_meta.num_children(); ++i) {
        add_default_name(col_meta.child(i), col_meta.get_name() + "_" + std::to_string(i));
      }
    };
  for (size_t i = 0; i < table_meta->column_metadata.size(); ++i) {
    add_default_name(table_meta->column_metadata[i], "_col" + std::to_string(i));
  }

  auto vec         = table_to_linked_columns(table);
  auto schema_tree = construct_schema_tree(vec, *table_meta, single_write_mode, int96_timestamps);
  // Construct parquet_column_views from the schema tree leaf nodes.
  std::vector<parquet_column_view> parquet_columns;

  for (schema_tree_node const& schema_node : schema_tree) {
    if (schema_node.leaf_column) { parquet_columns.emplace_back(schema_node, schema_tree, stream); }
  }

  // Mass allocation of column_device_views for each parquet_column_view
  std::vector<column_view> cudf_cols;
  cudf_cols.reserve(parquet_columns.size());
  for (auto const& parq_col : parquet_columns) {
    cudf_cols.push_back(parq_col.cudf_column_view());
  }
  table_view single_streams_table(cudf_cols);
  size_type num_columns = single_streams_table.num_columns();

  std::vector<SchemaElement> this_table_schema(schema_tree.begin(), schema_tree.end());

  if (!md) {
    md = std::make_unique<aggregate_writer_metadata>(
      partitions, num_columns, std::move(this_table_schema), stats_granularity_, kv_md);
  } else {
    // verify the user isn't passing mismatched tables
    CUDF_EXPECTS(md->schema_matches(this_table_schema),
                 "Mismatch in schema between multiple calls to write_chunk");

    md->update_files(partitions);
  }
  // Create table_device_view so that corresponding column_device_view data
  // can be written into col_desc members
  auto parent_column_table_device_view = table_device_view::create(single_streams_table, stream);
  rmm::device_uvector<column_device_view> leaf_column_views(0, stream);

  // Initialize column description
  hostdevice_vector<gpu::parquet_column_device_view> col_desc(parquet_columns.size(), stream);
  std::transform(
    parquet_columns.begin(), parquet_columns.end(), col_desc.host_ptr(), [&](auto const& pcol) {
      return pcol.get_device_view(stream);
    });

  // Init page fragments
  // 5000 is good enough for up to ~200-character strings. Longer strings will start producing
  // fragments larger than the desired page size -> TODO: keep track of the max fragment size, and
  // iteratively reduce this value if the largest fragment exceeds the max page size limit (we
  // ideally want the page size to be below 1MB so as to have enough pages to get good
  // compression/decompression performance).
  auto max_page_fragment_size =
    (cudf::io::parquet::gpu::max_page_fragment_size * max_page_size_bytes) /
    default_max_page_size_bytes;

  std::vector<int> num_frag_in_part;
  std::transform(partitions.begin(),
                 partitions.end(),
                 std::back_inserter(num_frag_in_part),
                 [max_page_fragment_size](auto const& part) {
                   return util::div_rounding_up_unsafe(part.num_rows, max_page_fragment_size);
                 });

  size_type num_fragments = std::reduce(num_frag_in_part.begin(), num_frag_in_part.end());

  std::vector<int> part_frag_offset;  // Store the idx of the first fragment in each partition
  std::exclusive_scan(
    num_frag_in_part.begin(), num_frag_in_part.end(), std::back_inserter(part_frag_offset), 0);
  part_frag_offset.push_back(part_frag_offset.back() + num_frag_in_part.back());

  auto d_part_frag_offset = cudf::detail::make_device_uvector_async(part_frag_offset, stream);
  cudf::detail::hostdevice_2dvector<gpu::PageFragment> fragments(
    num_columns, num_fragments, stream);

  if (num_fragments != 0) {
    // Move column info to device
    col_desc.host_to_device(stream);
    leaf_column_views = create_leaf_column_device_views<gpu::parquet_column_device_view>(
      col_desc, *parent_column_table_device_view, stream);

    init_page_fragments(
      fragments, col_desc, partitions, d_part_frag_offset, max_page_fragment_size);
  }

  std::vector<size_t> const global_rowgroup_base = md->num_row_groups_per_file();

  // Decide row group boundaries based on uncompressed data size
  int num_rowgroups = 0;

  std::vector<int> num_rg_in_part(partitions.size());
  for (size_t p = 0; p < partitions.size(); ++p) {
    size_type curr_rg_num_rows = 0;
    size_t curr_rg_data_size   = 0;
    int first_frag_in_rg       = part_frag_offset[p];
    int last_frag_in_part      = part_frag_offset[p + 1] - 1;
    for (auto f = first_frag_in_rg; f <= last_frag_in_part; ++f) {
      size_t fragment_data_size = 0;
      for (auto c = 0; c < num_columns; c++) {
        fragment_data_size += fragments[c][f].fragment_data_size;
      }
      size_type fragment_num_rows = fragments[0][f].num_rows;

      // If the fragment size gets larger than rg limit then break off a rg
      if (f > first_frag_in_rg &&  // There has to be at least one fragment in row group
          (curr_rg_data_size + fragment_data_size > max_row_group_size ||
           curr_rg_num_rows + fragment_num_rows > max_row_group_rows)) {
        auto& rg    = md->file(p).row_groups.emplace_back();
        rg.num_rows = curr_rg_num_rows;
        num_rowgroups++;
        num_rg_in_part[p]++;
        curr_rg_num_rows  = 0;
        curr_rg_data_size = 0;
        first_frag_in_rg  = f;
      }
      curr_rg_num_rows += fragment_num_rows;
      curr_rg_data_size += fragment_data_size;

      // TODO: (wishful) refactor to consolidate with above if block
      if (f == last_frag_in_part) {
        auto& rg    = md->file(p).row_groups.emplace_back();
        rg.num_rows = curr_rg_num_rows;
        num_rowgroups++;
        num_rg_in_part[p]++;
      }
    }
  }

  // Allocate column chunks and gather fragment statistics
  rmm::device_uvector<statistics_chunk> frag_stats(0, stream);
  if (stats_granularity_ != statistics_freq::STATISTICS_NONE) {
    frag_stats.resize(num_fragments * num_columns, stream);
    if (not frag_stats.is_empty()) {
      auto frag_stats_2dview =
        device_2dspan<statistics_chunk>(frag_stats.data(), num_columns, num_fragments);
      gather_fragment_statistics(frag_stats_2dview, fragments, col_desc, num_fragments);
    }
  }

  std::vector<int> first_rg_in_part;
  std::exclusive_scan(
    num_rg_in_part.begin(), num_rg_in_part.end(), std::back_inserter(first_rg_in_part), 0);

  // Initialize row groups and column chunks
  auto const num_chunks = num_rowgroups * num_columns;
  hostdevice_2dvector<gpu::EncColumnChunk> chunks(num_rowgroups, num_columns, stream);

  for (size_t p = 0; p < partitions.size(); ++p) {
    int f               = part_frag_offset[p];
    size_type start_row = partitions[p].start_row;
    for (int r = 0; r < num_rg_in_part[p]; r++) {
      size_t global_r = global_rowgroup_base[p] + r;  // Number of rowgroups already in file/part
      auto& row_group = md->file(p).row_groups[global_r];
      uint32_t fragments_in_chunk =
        util::div_rounding_up_unsafe(row_group.num_rows, max_page_fragment_size);
      row_group.total_byte_size = 0;
      row_group.columns.resize(num_columns);
      for (int c = 0; c < num_columns; c++) {
        gpu::EncColumnChunk& ck = chunks[r + first_rg_in_part[p]][c];

        ck             = {};
        ck.col_desc    = col_desc.device_ptr() + c;
        ck.col_desc_id = c;
        ck.fragments   = &fragments.device_view()[c][f];
        ck.stats =
          (not frag_stats.is_empty()) ? frag_stats.data() + c * num_fragments + f : nullptr;
        ck.start_row         = start_row;
        ck.num_rows          = (uint32_t)row_group.num_rows;
        ck.first_fragment    = c * num_fragments + f;
        auto chunk_fragments = fragments[c].subspan(f, fragments_in_chunk);
        // In fragment struct, add a pointer to the chunk it belongs to
        // In each fragment in chunk_fragments, update the chunk pointer here.
        for (auto& frag : chunk_fragments) {
          frag.chunk = &chunks.device_view()[r + first_rg_in_part[p]][c];
        }
        ck.num_values = std::accumulate(
          chunk_fragments.begin(), chunk_fragments.end(), 0, [](uint32_t l, auto r) {
            return l + r.num_values;
          });
        ck.plain_data_size = std::accumulate(
          chunk_fragments.begin(), chunk_fragments.end(), 0, [](int sum, gpu::PageFragment frag) {
            return sum + frag.fragment_data_size;
          });
        auto& column_chunk_meta          = row_group.columns[c].meta_data;
        column_chunk_meta.type           = parquet_columns[c].physical_type();
        column_chunk_meta.encodings      = {Encoding::PLAIN, Encoding::RLE};
        column_chunk_meta.path_in_schema = parquet_columns[c].get_path_in_schema();
        column_chunk_meta.codec          = UNCOMPRESSED;
        column_chunk_meta.num_values     = ck.num_values;
      }
      f += fragments_in_chunk;
      start_row += (uint32_t)row_group.num_rows;
    }
  }

  fragments.host_to_device(stream);
  auto dict_info_owner = build_chunk_dictionaries(chunks, col_desc, fragments, stream);
  for (size_t p = 0; p < partitions.size(); p++) {
    for (int rg = 0; rg < num_rg_in_part[p]; rg++) {
      size_t global_rg = global_rowgroup_base[p] + rg;
      for (int col = 0; col < num_columns; col++) {
        if (chunks.host_view()[rg][col].use_dictionary) {
          md->file(p).row_groups[global_rg].columns[col].meta_data.encodings.push_back(
            Encoding::PLAIN_DICTIONARY);
        }
      }
    }
  }

  // Build chunk dictionaries and count pages
  hostdevice_vector<size_type> comp_page_sizes = init_page_sizes(
    chunks, col_desc, num_columns, max_page_size_bytes, max_page_size_rows, compression_, stream);

  // Get the maximum page size across all chunks
  size_type max_page_uncomp_data_size =
    std::accumulate(chunks.host_view().flat_view().begin(),
                    chunks.host_view().flat_view().end(),
                    0,
                    [](uint32_t max_page_size, gpu::EncColumnChunk const& chunk) {
                      return std::max(max_page_size, chunk.max_page_data_size);
                    });

  // Find which partition a rg belongs to
  std::vector<int> rg_to_part;
  for (size_t p = 0; p < num_rg_in_part.size(); ++p) {
    std::fill_n(std::back_inserter(rg_to_part), num_rg_in_part[p], p);
  }

  // Initialize batches of rowgroups to encode (mainly to limit peak memory usage)
  std::vector<size_type> batch_list;
  size_type num_pages          = 0;
  size_t max_bytes_in_batch    = 1024 * 1024 * 1024;  // 1GB - TODO: Tune this
  size_t max_uncomp_bfr_size   = 0;
  size_t max_comp_bfr_size     = 0;
  size_t max_chunk_bfr_size    = 0;
  size_type max_pages_in_batch = 0;
  size_t bytes_in_batch        = 0;
  size_t comp_bytes_in_batch   = 0;
  size_t column_index_bfr_size = 0;
  for (size_type r = 0, groups_in_batch = 0, pages_in_batch = 0; r <= num_rowgroups; r++) {
    size_t rowgroup_size      = 0;
    size_t comp_rowgroup_size = 0;
    if (r < num_rowgroups) {
      for (int i = 0; i < num_columns; i++) {
        gpu::EncColumnChunk* ck = &chunks[r][i];
        ck->first_page          = num_pages;
        num_pages += ck->num_pages;
        pages_in_batch += ck->num_pages;
        rowgroup_size += ck->bfr_size;
        comp_rowgroup_size += ck->compressed_size;
        max_chunk_bfr_size =
          std::max(max_chunk_bfr_size, (size_t)std::max(ck->bfr_size, ck->compressed_size));
        if (stats_granularity_ == statistics_freq::STATISTICS_COLUMN) {
          column_index_bfr_size += column_index_buffer_size(ck);
        }
      }
    }
    // TBD: We may want to also shorten the batch if we have enough pages (not just based on size)
    if ((r == num_rowgroups) ||
        (groups_in_batch != 0 && bytes_in_batch + rowgroup_size > max_bytes_in_batch)) {
      max_uncomp_bfr_size = std::max(max_uncomp_bfr_size, bytes_in_batch);
      max_comp_bfr_size   = std::max(max_comp_bfr_size, comp_bytes_in_batch);
      max_pages_in_batch  = std::max(max_pages_in_batch, pages_in_batch);
      if (groups_in_batch != 0) {
        batch_list.push_back(groups_in_batch);
        groups_in_batch = 0;
      }
      bytes_in_batch      = 0;
      comp_bytes_in_batch = 0;
      pages_in_batch      = 0;
    }
    bytes_in_batch += rowgroup_size;
    comp_bytes_in_batch += comp_rowgroup_size;
    groups_in_batch++;
  }

  // Clear compressed buffer size if compression has been turned off
  if (compression_ == parquet::Compression::UNCOMPRESSED) { max_comp_bfr_size = 0; }

  // Initialize data pointers in batch
  uint32_t num_stats_bfr =
    (stats_granularity_ != statistics_freq::STATISTICS_NONE) ? num_pages + num_chunks : 0;
  rmm::device_buffer uncomp_bfr(max_uncomp_bfr_size, stream);
  rmm::device_buffer comp_bfr(max_comp_bfr_size, stream);
  rmm::device_buffer col_idx_bfr(column_index_bfr_size, stream);
  rmm::device_uvector<gpu::EncPage> pages(num_pages, stream);

  // This contains stats for both the pages and the rowgroups. TODO: make them separate.
  rmm::device_uvector<statistics_chunk> page_stats(num_stats_bfr, stream);
  auto bfr_i = static_cast<uint8_t*>(col_idx_bfr.data());
  for (auto b = 0, r = 0; b < static_cast<size_type>(batch_list.size()); b++) {
    auto bfr   = static_cast<uint8_t*>(uncomp_bfr.data());
    auto bfr_c = static_cast<uint8_t*>(comp_bfr.data());
    for (auto j = 0; j < batch_list[b]; j++, r++) {
      for (auto i = 0; i < num_columns; i++) {
        gpu::EncColumnChunk& ck = chunks[r][i];
        ck.uncompressed_bfr     = bfr;
        ck.compressed_bfr       = bfr_c;
        ck.column_index_blob    = bfr_i;
        bfr += ck.bfr_size;
        bfr_c += ck.compressed_size;
        if (stats_granularity_ == statistics_freq::STATISTICS_COLUMN) {
          ck.column_index_size = column_index_buffer_size(&ck);
          bfr_i += ck.column_index_size;
        }
      }
    }
  }

  if (num_pages != 0) {
    init_encoder_pages(chunks,
                       col_desc,
                       {pages.data(), pages.size()},
                       comp_page_sizes,
                       (num_stats_bfr) ? page_stats.data() : nullptr,
                       (num_stats_bfr) ? frag_stats.data() : nullptr,
                       num_columns,
                       num_pages,
                       num_stats_bfr);
  }

  pinned_buffer<uint8_t> host_bfr{nullptr, hipHostFree};

  // Encode row groups in batches
  for (auto b = 0, r = 0; b < static_cast<size_type>(batch_list.size()); b++) {
    // Count pages in this batch
    auto const rnext               = r + batch_list[b];
    auto const first_page_in_batch = chunks[r][0].first_page;
    auto const first_page_in_next_batch =
      (rnext < num_rowgroups) ? chunks[rnext][0].first_page : num_pages;
    auto const pages_in_batch = first_page_in_next_batch - first_page_in_batch;
    encode_pages(
      chunks,
      {pages.data(), pages.size()},
      max_page_uncomp_data_size,
      pages_in_batch,
      first_page_in_batch,
      batch_list[b],
      r,
      (stats_granularity_ == statistics_freq::STATISTICS_PAGE) ? page_stats.data() : nullptr,
      (stats_granularity_ != statistics_freq::STATISTICS_NONE) ? page_stats.data() + num_pages
                                                               : nullptr,
      (stats_granularity_ == statistics_freq::STATISTICS_COLUMN) ? page_stats.data() : nullptr);

    std::vector<std::future<void>> write_tasks;
    for (; r < rnext; r++) {
      int p           = rg_to_part[r];
      int global_r    = global_rowgroup_base[p] + r - first_rg_in_part[p];
      auto& row_group = md->file(p).row_groups[global_r];
      for (auto i = 0; i < num_columns; i++) {
        gpu::EncColumnChunk& ck = chunks[r][i];
        auto& column_chunk_meta = row_group.columns[i].meta_data;
        uint8_t* dev_bfr;
        if (ck.is_compressed) {
          column_chunk_meta.codec = compression_;
          dev_bfr                 = ck.compressed_bfr;
        } else {
          dev_bfr = ck.uncompressed_bfr;
        }

        if (out_sink_[p]->is_device_write_preferred(ck.compressed_size)) {
          // let the writer do what it wants to retrieve the data from the gpu.
          write_tasks.push_back(out_sink_[p]->device_write_async(
            dev_bfr + ck.ck_stat_size, ck.compressed_size, stream));
          // we still need to do a (much smaller) memcpy for the statistics.
          if (ck.ck_stat_size != 0) {
            column_chunk_meta.statistics_blob.resize(ck.ck_stat_size);
            CUDF_CUDA_TRY(hipMemcpyAsync(column_chunk_meta.statistics_blob.data(),
                                          dev_bfr,
                                          ck.ck_stat_size,
                                          hipMemcpyDeviceToHost,
                                          stream.value()));
            stream.synchronize();
          }
        } else {
          if (!host_bfr) {
            host_bfr = pinned_buffer<uint8_t>{[](size_t size) {
                                                uint8_t* ptr = nullptr;
                                                CUDF_CUDA_TRY(hipHostMalloc(&ptr, size));
                                                return ptr;
                                              }(max_chunk_bfr_size),
                                              hipHostFree};
          }
          // copy the full data
          CUDF_CUDA_TRY(hipMemcpyAsync(host_bfr.get(),
                                        dev_bfr,
                                        ck.ck_stat_size + ck.compressed_size,
                                        hipMemcpyDeviceToHost,
                                        stream.value()));
          stream.synchronize();
          out_sink_[p]->host_write(host_bfr.get() + ck.ck_stat_size, ck.compressed_size);
          if (ck.ck_stat_size != 0) {
            column_chunk_meta.statistics_blob.resize(ck.ck_stat_size);
            memcpy(column_chunk_meta.statistics_blob.data(), host_bfr.get(), ck.ck_stat_size);
          }
        }
        row_group.total_byte_size += ck.compressed_size;
        column_chunk_meta.data_page_offset =
          current_chunk_offset[p] + ((ck.use_dictionary) ? ck.dictionary_size : 0);
        column_chunk_meta.dictionary_page_offset =
          (ck.use_dictionary) ? current_chunk_offset[p] : 0;
        column_chunk_meta.total_uncompressed_size = ck.bfr_size;
        column_chunk_meta.total_compressed_size   = ck.compressed_size;
        current_chunk_offset[p] += ck.compressed_size;
      }
    }
    for (auto const& task : write_tasks) {
      task.wait();
    }
  }

  if (stats_granularity_ == statistics_freq::STATISTICS_COLUMN) {
    // need pages on host to create offset_indexes
    thrust::host_vector<gpu::EncPage> h_pages = cudf::detail::make_host_vector_async(pages, stream);
    stream.synchronize();

    // add column and offset indexes to metadata
    for (auto b = 0, r = 0; b < static_cast<size_type>(batch_list.size()); b++) {
      auto const rnext   = r + batch_list[b];
      auto curr_page_idx = chunks[r][0].first_page;
      for (; r < rnext; r++) {
        int p                 = rg_to_part[r];
        int global_r          = global_rowgroup_base[p] + r - first_rg_in_part[p];
        auto const& row_group = md->file(p).row_groups[global_r];
        for (auto i = 0; i < num_columns; i++) {
          gpu::EncColumnChunk const& ck = chunks[r][i];
          auto const& column_chunk_meta = row_group.columns[i].meta_data;

          // start transfer of the column index
          std::vector<uint8_t> column_idx;
          column_idx.resize(ck.column_index_size);
          CUDF_CUDA_TRY(hipMemcpyAsync(column_idx.data(),
                                        ck.column_index_blob,
                                        ck.column_index_size,
                                        hipMemcpyDeviceToHost,
                                        stream.value()));

          // calculate offsets while the column index is transferring
          int64_t curr_pg_offset = column_chunk_meta.data_page_offset;

          OffsetIndex offset_idx;
          for (uint32_t pg = 0; pg < ck.num_pages; pg++) {
            auto const& enc_page = h_pages[curr_page_idx++];

            // skip dict pages
            if (enc_page.page_type != PageType::DATA_PAGE) { continue; }

            int32_t this_page_size = enc_page.hdr_size + enc_page.max_data_size;
            // first_row_idx is relative to start of row group
            PageLocation loc{curr_pg_offset, this_page_size, enc_page.start_row - ck.start_row};
            offset_idx.page_locations.push_back(loc);
            curr_pg_offset += this_page_size;
          }

          stream.synchronize();
          md->file(p).offset_indexes.push_back(offset_idx);
          md->file(p).column_indexes.push_back(column_idx);
        }
      }
    }
  }

  last_write_successful = true;
}

std::unique_ptr<std::vector<uint8_t>> writer::impl::close(
  std::vector<std::string> const& column_chunks_file_path)
{
  if (closed) { return nullptr; }
  closed = true;
  if (not last_write_successful) { return nullptr; }
  for (size_t p = 0; p < out_sink_.size(); p++) {
    std::vector<uint8_t> buffer;
    CompactProtocolWriter cpw(&buffer);
    file_ender_s fendr;

    if (stats_granularity_ == statistics_freq::STATISTICS_COLUMN) {
      auto& fmd = md->file(p);

      // write column indices, updating column metadata along the way
      int chunkidx = 0;
      for (auto& r : fmd.row_groups) {
        for (auto& c : r.columns) {
          auto const& index     = fmd.column_indexes[chunkidx++];
          c.column_index_offset = out_sink_[p]->bytes_written();
          c.column_index_length = index.size();
          out_sink_[p]->host_write(index.data(), index.size());
        }
      }

      // write offset indices, updating column metadata along the way
      chunkidx = 0;
      for (auto& r : fmd.row_groups) {
        for (auto& c : r.columns) {
          auto const& offsets = fmd.offset_indexes[chunkidx++];
          buffer.resize(0);
          int32_t len           = cpw.write(offsets);
          c.offset_index_offset = out_sink_[p]->bytes_written();
          c.offset_index_length = len;
          out_sink_[p]->host_write(buffer.data(), buffer.size());
        }
      }
    }

    buffer.resize(0);
    fendr.footer_len = static_cast<uint32_t>(cpw.write(md->get_metadata(p)));
    fendr.magic      = parquet_magic;
    out_sink_[p]->host_write(buffer.data(), buffer.size());
    out_sink_[p]->host_write(&fendr, sizeof(fendr));
    out_sink_[p]->flush();
  }

  // Optionally output raw file metadata with the specified column chunk file path
  if (column_chunks_file_path.size() > 0) {
    CUDF_EXPECTS(column_chunks_file_path.size() == md->num_files(),
                 "Expected one column chunk path per output file");
    md->set_file_paths(column_chunks_file_path);
    file_header_s fhdr = {parquet_magic};
    std::vector<uint8_t> buffer;
    CompactProtocolWriter cpw(&buffer);
    buffer.insert(buffer.end(),
                  reinterpret_cast<const uint8_t*>(&fhdr),
                  reinterpret_cast<const uint8_t*>(&fhdr) + sizeof(fhdr));
    file_ender_s fendr;
    fendr.magic      = parquet_magic;
    fendr.footer_len = static_cast<uint32_t>(cpw.write(md->get_merged_metadata()));
    buffer.insert(buffer.end(),
                  reinterpret_cast<const uint8_t*>(&fendr),
                  reinterpret_cast<const uint8_t*>(&fendr) + sizeof(fendr));
    return std::make_unique<std::vector<uint8_t>>(std::move(buffer));
  } else {
    return {nullptr};
  }
  return nullptr;
}

// Forward to implementation
writer::writer(std::vector<std::unique_ptr<data_sink>> sinks,
               parquet_writer_options const& options,
               SingleWriteMode mode,
               rmm::cuda_stream_view stream,
               rmm::mr::device_memory_resource* mr)
  : _impl(std::make_unique<impl>(std::move(sinks), options, mode, stream, mr))
{
}

writer::writer(std::vector<std::unique_ptr<data_sink>> sinks,
               chunked_parquet_writer_options const& options,
               SingleWriteMode mode,
               rmm::cuda_stream_view stream,
               rmm::mr::device_memory_resource* mr)
  : _impl(std::make_unique<impl>(std::move(sinks), options, mode, stream, mr))
{
}

// Destructor within this translation unit
writer::~writer() = default;

// Forward to implementation
void writer::write(table_view const& table, std::vector<partition_info> const& partitions)
{
  _impl->write(
    table, partitions.empty() ? std::vector<partition_info>{{0, table.num_rows()}} : partitions);
}

// Forward to implementation
std::unique_ptr<std::vector<uint8_t>> writer::close(
  std::vector<std::string> const& column_chunks_file_path)
{
  return _impl->close(column_chunks_file_path);
}

std::unique_ptr<std::vector<uint8_t>> writer::merge_row_group_metadata(
  std::vector<std::unique_ptr<std::vector<uint8_t>>> const& metadata_list)
{
  std::vector<uint8_t> output;
  CompactProtocolWriter cpw(&output);
  FileMetaData md;

  md.row_groups.reserve(metadata_list.size());
  for (const auto& blob : metadata_list) {
    CompactProtocolReader cpreader(
      blob.get()->data(),
      std::max<size_t>(blob.get()->size(), sizeof(file_ender_s)) - sizeof(file_ender_s));
    cpreader.skip_bytes(sizeof(file_header_s));  // Skip over file header
    if (md.num_rows == 0) {
      cpreader.read(&md);
    } else {
      FileMetaData tmp;
      cpreader.read(&tmp);
      md.row_groups.insert(md.row_groups.end(),
                           std::make_move_iterator(tmp.row_groups.begin()),
                           std::make_move_iterator(tmp.row_groups.end()));
      md.num_rows += tmp.num_rows;
    }
  }
  // Reader doesn't currently populate column_order, so infer it here
  if (md.row_groups.size() != 0) {
    uint32_t num_columns = static_cast<uint32_t>(md.row_groups[0].columns.size());
    md.column_order_listsize =
      (num_columns > 0 && md.row_groups[0].columns[0].meta_data.statistics_blob.size())
        ? num_columns
        : 0;
  }
  // Thrift-encode the resulting output
  file_header_s fhdr;
  file_ender_s fendr;
  fhdr.magic = parquet_magic;
  output.insert(output.end(),
                reinterpret_cast<const uint8_t*>(&fhdr),
                reinterpret_cast<const uint8_t*>(&fhdr) + sizeof(fhdr));
  fendr.footer_len = static_cast<uint32_t>(cpw.write(md));
  fendr.magic      = parquet_magic;
  output.insert(output.end(),
                reinterpret_cast<const uint8_t*>(&fendr),
                reinterpret_cast<const uint8_t*>(&fendr) + sizeof(fendr));
  return std::make_unique<std::vector<uint8_t>>(std::move(output));
}

}  // namespace parquet
}  // namespace detail
}  // namespace io
}  // namespace cudf
