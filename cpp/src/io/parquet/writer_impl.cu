#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019-2024, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

/**
 * @file writer_impl.cu
 * @brief cuDF-IO parquet writer class implementation
 */

#include "compact_protocol_reader.hpp"
#include "compact_protocol_writer.hpp"
#include "io/comp/nvcomp_adapter.hpp"
#include "io/parquet/parquet.hpp"
#include "io/parquet/parquet_gpu.hpp"
#include "io/statistics/column_statistics.cuh"
#include "io/utilities/column_utils.cuh"
#include "io/utilities/config_utils.hpp"
#include "parquet_common.hpp"
#include "parquet_gpu.cuh"
#include "writer_impl.hpp"

#include <cudf/column/column_device_view.cuh>
#include <cudf/copying.hpp>
#include <cudf/detail/get_value.cuh>
#include <cudf/detail/utilities/integer_utils.hpp>
#include <cudf/detail/utilities/linked_column.hpp>
#include <cudf/detail/utilities/pinned_host_vector.hpp>
#include <cudf/detail/utilities/vector_factories.hpp>
#include <cudf/lists/detail/dremel.hpp>
#include <cudf/lists/lists_column_view.hpp>
#include <cudf/strings/detail/utilities.hpp>
#include <cudf/strings/strings_column_view.hpp>
#include <cudf/structs/structs_column_view.hpp>
#include <cudf/table/table_device_view.cuh>

#include <rmm/cuda_stream_view.hpp>
#include <rmm/device_buffer.hpp>
#include <rmm/device_uvector.hpp>

#include <thrust/fill.h>
#include <thrust/for_each.h>

#include <algorithm>
#include <cstring>
#include <iterator>
#include <numeric>
#include <utility>

#ifndef CUDF_VERSION
#error "CUDF_VERSION is not defined"
#endif

namespace cudf::io::parquet::detail {

using namespace cudf::io::detail;

struct aggregate_writer_metadata {
  aggregate_writer_metadata(host_span<partition_info const> partitions,
                            host_span<std::map<std::string, std::string> const> kv_md,
                            host_span<SchemaElement const> tbl_schema,
                            size_type num_columns,
                            statistics_freq stats_granularity)
    : version(1),
      schema(std::vector<SchemaElement>(tbl_schema.begin(), tbl_schema.end())),
      files(partitions.size())
  {
    for (size_t i = 0; i < partitions.size(); ++i) {
      this->files[i].num_rows = partitions[i].num_rows;
    }

    if (stats_granularity != statistics_freq::STATISTICS_NONE) {
      ColumnOrder default_order = {ColumnOrder::TYPE_ORDER};
      this->column_orders       = std::vector<ColumnOrder>(num_columns, default_order);
    }

    for (size_t p = 0; p < kv_md.size(); ++p) {
      std::transform(kv_md[p].begin(),
                     kv_md[p].end(),
                     std::back_inserter(this->files[p].key_value_metadata),
                     [](auto const& kv) {
                       return KeyValue{kv.first, kv.second};
                     });
    }
  }

  aggregate_writer_metadata(aggregate_writer_metadata const&) = default;

  void update_files(host_span<partition_info const> partitions)
  {
    CUDF_EXPECTS(partitions.size() == this->files.size(),
                 "New partitions must be same size as previously passed number of partitions");
    for (size_t i = 0; i < partitions.size(); ++i) {
      this->files[i].num_rows += partitions[i].num_rows;
    }
  }

  FileMetaData get_metadata(size_t part)
  {
    CUDF_EXPECTS(part < files.size(), "Invalid part index queried");
    FileMetaData meta{};
    meta.version            = this->version;
    meta.schema             = this->schema;
    meta.num_rows           = this->files[part].num_rows;
    meta.row_groups         = this->files[part].row_groups;
    meta.key_value_metadata = this->files[part].key_value_metadata;
    meta.created_by         = "cudf version " CUDF_STRINGIFY(CUDF_VERSION);
    meta.column_orders      = this->column_orders;
    return meta;
  }

  void set_file_paths(host_span<std::string const> column_chunks_file_path)
  {
    for (size_t p = 0; p < this->files.size(); ++p) {
      auto& file            = this->files[p];
      auto const& file_path = column_chunks_file_path[p];
      for (auto& rowgroup : file.row_groups) {
        for (auto& col : rowgroup.columns) {
          col.file_path = file_path;
        }
      }
    }
  }

  FileMetaData get_merged_metadata()
  {
    FileMetaData merged_md;
    for (size_t p = 0; p < this->files.size(); ++p) {
      auto& file = this->files[p];
      if (p == 0) {
        merged_md = this->get_metadata(0);
      } else {
        merged_md.row_groups.insert(merged_md.row_groups.end(),
                                    std::make_move_iterator(file.row_groups.begin()),
                                    std::make_move_iterator(file.row_groups.end()));
        merged_md.num_rows += file.num_rows;
      }
    }
    return merged_md;
  }

  std::vector<size_t> num_row_groups_per_file()
  {
    std::vector<size_t> global_rowgroup_base;
    std::transform(this->files.begin(),
                   this->files.end(),
                   std::back_inserter(global_rowgroup_base),
                   [](auto const& part) { return part.row_groups.size(); });
    return global_rowgroup_base;
  }

  [[nodiscard]] bool schema_matches(std::vector<SchemaElement> const& schema) const
  {
    return this->schema == schema;
  }
  auto& file(size_t p) { return files[p]; }
  [[nodiscard]] size_t num_files() const { return files.size(); }

 private:
  int32_t version = 0;
  std::vector<SchemaElement> schema;
  struct per_file_metadata {
    int64_t num_rows = 0;
    std::vector<RowGroup> row_groups;
    std::vector<KeyValue> key_value_metadata;
    std::vector<OffsetIndex> offset_indexes;
    std::vector<std::vector<uint8_t>> column_indexes;
  };
  std::vector<per_file_metadata> files;
  thrust::optional<std::vector<ColumnOrder>> column_orders = thrust::nullopt;
};

namespace {

/**
 * @brief Function that translates GDF compression to parquet compression.
 *
 * @param compression The compression type
 * @return The supported Parquet compression
 */
Compression to_parquet_compression(compression_type compression)
{
  switch (compression) {
    case compression_type::AUTO:
    case compression_type::SNAPPY: return Compression::SNAPPY;
    case compression_type::ZSTD: return Compression::ZSTD;
    case compression_type::LZ4:
      // Parquet refers to LZ4 as "LZ4_RAW"; Parquet's "LZ4" is not standard LZ4
      return Compression::LZ4_RAW;
    case compression_type::NONE: return Compression::UNCOMPRESSED;
    default: CUDF_FAIL("Unsupported compression type");
  }
}

/**
 * @brief Convert a mask of encodings to a vector.
 *
 * @param encodings Vector of `Encoding`s to populate
 * @param enc_mask Mask of encodings used
 */
void update_chunk_encodings(std::vector<Encoding>& encodings, uint32_t enc_mask)
{
  for (uint8_t enc = 0; enc < static_cast<uint8_t>(Encoding::NUM_ENCODINGS); enc++) {
    auto const enc_enum = static_cast<Encoding>(enc);
    if ((enc_mask & encoding_to_mask(enc_enum)) != 0) { encodings.push_back(enc_enum); }
  }
}

/**
 * @brief Update the encoding_stats field in the column chunk metadata.
 *
 * @param chunk_meta The `ColumnChunkMetaData` struct for the column chunk
 * @param ck The column chunk to summarize stats for
 * @param is_v2 True if V2 page headers are used
 */
void update_chunk_encoding_stats(ColumnChunkMetaData& chunk_meta,
                                 EncColumnChunk const& ck,
                                 bool is_v2)
{
  // don't set encoding stats if there are no pages
  if (ck.num_pages == 0) { return; }

  // NOTE: since cudf doesn't use mixed encodings for a chunk, we really only need to account
  // for the dictionary page (if there is one), and the encoding used for the data pages. We can
  // examine the chunk's encodings field to figure out the encodings without having to examine
  // the page data.
  auto const num_data_pages = static_cast<int32_t>(ck.num_data_pages());
  auto const data_page_type = is_v2 ? PageType::DATA_PAGE_V2 : PageType::DATA_PAGE;

  std::vector<PageEncodingStats> result;
  if (ck.use_dictionary) {
    // For dictionary encoding, if V1 then both data and dictionary use PLAIN_DICTIONARY. For V2
    // the dictionary uses PLAIN and the data RLE_DICTIONARY.
    auto const dict_enc = is_v2 ? Encoding::PLAIN : Encoding::PLAIN_DICTIONARY;
    auto const data_enc = is_v2 ? Encoding::RLE_DICTIONARY : Encoding::PLAIN_DICTIONARY;
    result.push_back({PageType::DICTIONARY_PAGE, dict_enc, 1});
    if (num_data_pages > 0) { result.push_back({data_page_type, data_enc, num_data_pages}); }
  } else {
    // No dictionary page, the pages are encoded with something other than RLE (unless it's a
    // boolean column).
    for (auto const enc : chunk_meta.encodings) {
      if (enc != Encoding::RLE) {
        result.push_back({data_page_type, enc, num_data_pages});
        break;
      }
    }
    // if result is empty and we're using V2 headers, then assume the data is RLE as well
    if (result.empty() and is_v2 and (ck.encodings & encoding_to_mask(Encoding::RLE)) != 0) {
      result.push_back({data_page_type, Encoding::RLE, num_data_pages});
    }
  }

  if (not result.empty()) { chunk_meta.encoding_stats = std::move(result); }
}

/**
 * @brief Compute size (in bytes) of the data stored in the given column.
 *
 * @param column The input column
 * @param stream CUDA stream used for device memory operations and kernel launches
 * @return The data size of the input
 */
size_t column_size(column_view const& column, rmm::cuda_stream_view stream)
{
  if (column.is_empty()) { return 0; }

  if (is_fixed_width(column.type())) {
    return size_of(column.type()) * column.size();
  } else if (column.type().id() == type_id::STRING) {
    auto const scol = strings_column_view(column);
    return cudf::strings::detail::get_offset_value(
             scol.offsets(), column.size() + column.offset(), stream) -
           cudf::strings::detail::get_offset_value(scol.offsets(), column.offset(), stream);
  } else if (column.type().id() == type_id::STRUCT) {
    auto const scol = structs_column_view(column);
    size_t ret      = 0;
    for (int i = 0; i < scol.num_children(); i++) {
      ret += column_size(scol.get_sliced_child(i, stream), stream);
    }
    return ret;
  } else if (column.type().id() == type_id::LIST) {
    auto const lcol = lists_column_view(column);
    return column_size(lcol.get_sliced_child(stream), stream);
  }

  CUDF_FAIL("Unexpected compound type");
}

// checks to see if the given column has a fixed size.  This doesn't
// check every row, so assumes string and list columns are not fixed, even
// if each row is the same width.
// TODO: update this if FIXED_LEN_BYTE_ARRAY is ever supported for writes.
bool is_col_fixed_width(column_view const& column)
{
  if (column.type().id() == type_id::STRUCT) {
    return std::all_of(column.child_begin(), column.child_end(), is_col_fixed_width);
  }

  return is_fixed_width(column.type());
}

/**
 * @brief Extends SchemaElement to add members required in constructing parquet_column_view
 *
 * Added members are:
 * 1. leaf_column: Pointer to leaf linked_column_view which points to the corresponding data stream
 *    of a leaf schema node. For non-leaf struct node, this is nullptr.
 * 2. stats_dtype: datatype for statistics calculation required for the data stream of a leaf node.
 * 3. ts_scale: scale to multiply or divide timestamp by in order to convert timestamp to parquet
 *    supported types
 * 4. requested_encoding: A user provided encoding to use for the column.
 */
struct schema_tree_node : public SchemaElement {
  cudf::detail::LinkedColPtr leaf_column;
  statistics_dtype stats_dtype;
  int32_t ts_scale;
  column_encoding requested_encoding;
  bool skip_compression;

  // TODO(fut): Think about making schema a class that holds a vector of schema_tree_nodes. The
  // function construct_schema_tree could be its constructor. It can have method to get the per
  // column nullability given a schema node index corresponding to a leaf schema. Much easier than
  // that is a method to get path in schema, given a leaf node
};

struct leaf_schema_fn {
  schema_tree_node& col_schema;
  cudf::detail::LinkedColPtr const& col;
  column_in_metadata const& col_meta;
  bool timestamp_is_int96;
  bool timestamp_is_utc;

  template <typename T>
  std::enable_if_t<std::is_same_v<T, bool>, void> operator()()
  {
    col_schema.type        = Type::BOOLEAN;
    col_schema.stats_dtype = statistics_dtype::dtype_bool;
    // BOOLEAN needs no converted or logical type
  }

  template <typename T>
  std::enable_if_t<std::is_same_v<T, int8_t>, void> operator()()
  {
    col_schema.type           = Type::INT32;
    col_schema.converted_type = ConvertedType::INT_8;
    col_schema.stats_dtype    = statistics_dtype::dtype_int8;
    col_schema.logical_type   = LogicalType{IntType{8, true}};
  }

  template <typename T>
  std::enable_if_t<std::is_same_v<T, int16_t>, void> operator()()
  {
    col_schema.type           = Type::INT32;
    col_schema.converted_type = ConvertedType::INT_16;
    col_schema.stats_dtype    = statistics_dtype::dtype_int16;
    col_schema.logical_type   = LogicalType{IntType{16, true}};
  }

  template <typename T>
  std::enable_if_t<std::is_same_v<T, int32_t>, void> operator()()
  {
    col_schema.type        = Type::INT32;
    col_schema.stats_dtype = statistics_dtype::dtype_int32;
    // INT32 needs no converted or logical type
  }

  template <typename T>
  std::enable_if_t<std::is_same_v<T, int64_t>, void> operator()()
  {
    col_schema.type        = Type::INT64;
    col_schema.stats_dtype = statistics_dtype::dtype_int64;
    // INT64 needs no converted or logical type
  }

  template <typename T>
  std::enable_if_t<std::is_same_v<T, uint8_t>, void> operator()()
  {
    col_schema.type           = Type::INT32;
    col_schema.converted_type = ConvertedType::UINT_8;
    col_schema.stats_dtype    = statistics_dtype::dtype_int8;
    col_schema.logical_type   = LogicalType{IntType{8, false}};
  }

  template <typename T>
  std::enable_if_t<std::is_same_v<T, uint16_t>, void> operator()()
  {
    col_schema.type           = Type::INT32;
    col_schema.converted_type = ConvertedType::UINT_16;
    col_schema.stats_dtype    = statistics_dtype::dtype_int16;
    col_schema.logical_type   = LogicalType{IntType{16, false}};
  }

  template <typename T>
  std::enable_if_t<std::is_same_v<T, uint32_t>, void> operator()()
  {
    col_schema.type           = Type::INT32;
    col_schema.converted_type = ConvertedType::UINT_32;
    col_schema.stats_dtype    = statistics_dtype::dtype_int32;
    col_schema.logical_type   = LogicalType{IntType{32, false}};
  }

  template <typename T>
  std::enable_if_t<std::is_same_v<T, uint64_t>, void> operator()()
  {
    col_schema.type           = Type::INT64;
    col_schema.converted_type = ConvertedType::UINT_64;
    col_schema.stats_dtype    = statistics_dtype::dtype_int64;
    col_schema.logical_type   = LogicalType{IntType{64, false}};
  }

  template <typename T>
  std::enable_if_t<std::is_same_v<T, float>, void> operator()()
  {
    col_schema.type        = Type::FLOAT;
    col_schema.stats_dtype = statistics_dtype::dtype_float32;
    // FLOAT needs no converted or logical type
  }

  template <typename T>
  std::enable_if_t<std::is_same_v<T, double>, void> operator()()
  {
    col_schema.type        = Type::DOUBLE;
    col_schema.stats_dtype = statistics_dtype::dtype_float64;
    // DOUBLE needs no converted or logical type
  }

  template <typename T>
  std::enable_if_t<std::is_same_v<T, cudf::string_view>, void> operator()()
  {
    col_schema.type = Type::BYTE_ARRAY;
    if (col_meta.is_enabled_output_as_binary()) {
      col_schema.stats_dtype = statistics_dtype::dtype_byte_array;
      // BYTE_ARRAY needs no converted or logical type
    } else {
      col_schema.converted_type = ConvertedType::UTF8;
      col_schema.stats_dtype    = statistics_dtype::dtype_string;
      col_schema.logical_type   = LogicalType{LogicalType::STRING};
    }
  }

  template <typename T>
  std::enable_if_t<std::is_same_v<T, cudf::timestamp_D>, void> operator()()
  {
    col_schema.type           = Type::INT32;
    col_schema.converted_type = ConvertedType::DATE;
    col_schema.stats_dtype    = statistics_dtype::dtype_int32;
    col_schema.logical_type   = LogicalType{LogicalType::DATE};
  }

  template <typename T>
  std::enable_if_t<std::is_same_v<T, cudf::timestamp_s>, void> operator()()
  {
    col_schema.type        = (timestamp_is_int96) ? Type::INT96 : Type::INT64;
    col_schema.stats_dtype = statistics_dtype::dtype_timestamp64;
    col_schema.ts_scale    = 1000;
    if (not timestamp_is_int96) {
      col_schema.converted_type = ConvertedType::TIMESTAMP_MILLIS;
      col_schema.logical_type   = LogicalType{TimestampType{timestamp_is_utc, TimeUnit::MILLIS}};
    }
  }

  template <typename T>
  std::enable_if_t<std::is_same_v<T, cudf::timestamp_ms>, void> operator()()
  {
    col_schema.type        = (timestamp_is_int96) ? Type::INT96 : Type::INT64;
    col_schema.stats_dtype = statistics_dtype::dtype_timestamp64;
    if (not timestamp_is_int96) {
      col_schema.converted_type = ConvertedType::TIMESTAMP_MILLIS;
      col_schema.logical_type   = LogicalType{TimestampType{timestamp_is_utc, TimeUnit::MILLIS}};
    }
  }

  template <typename T>
  std::enable_if_t<std::is_same_v<T, cudf::timestamp_us>, void> operator()()
  {
    col_schema.type        = (timestamp_is_int96) ? Type::INT96 : Type::INT64;
    col_schema.stats_dtype = statistics_dtype::dtype_timestamp64;
    if (not timestamp_is_int96) {
      col_schema.converted_type = ConvertedType::TIMESTAMP_MICROS;
      col_schema.logical_type   = LogicalType{TimestampType{timestamp_is_utc, TimeUnit::MICROS}};
    }
  }

  template <typename T>
  std::enable_if_t<std::is_same_v<T, cudf::timestamp_ns>, void> operator()()
  {
    col_schema.type           = (timestamp_is_int96) ? Type::INT96 : Type::INT64;
    col_schema.converted_type = thrust::nullopt;
    col_schema.stats_dtype    = statistics_dtype::dtype_timestamp64;
    if (timestamp_is_int96) {
      col_schema.ts_scale = -1000;  // negative value indicates division by absolute value
    }
    // set logical type if it's not int96
    else {
      col_schema.logical_type = LogicalType{TimestampType{timestamp_is_utc, TimeUnit::NANOS}};
    }
  }

  //  unsupported outside cudf for parquet 1.0.
  template <typename T>
  std::enable_if_t<std::is_same_v<T, cudf::duration_D>, void> operator()()
  {
    col_schema.type           = Type::INT32;
    col_schema.converted_type = ConvertedType::TIME_MILLIS;
    col_schema.stats_dtype    = statistics_dtype::dtype_int32;
    col_schema.ts_scale       = 24 * 60 * 60 * 1000;
    col_schema.logical_type   = LogicalType{TimeType{timestamp_is_utc, TimeUnit::MILLIS}};
  }

  template <typename T>
  std::enable_if_t<std::is_same_v<T, cudf::duration_s>, void> operator()()
  {
    col_schema.type           = Type::INT32;
    col_schema.converted_type = ConvertedType::TIME_MILLIS;
    col_schema.stats_dtype    = statistics_dtype::dtype_int32;
    col_schema.ts_scale       = 1000;
    col_schema.logical_type   = LogicalType{TimeType{timestamp_is_utc, TimeUnit::MILLIS}};
  }

  template <typename T>
  std::enable_if_t<std::is_same_v<T, cudf::duration_ms>, void> operator()()
  {
    col_schema.type           = Type::INT32;
    col_schema.converted_type = ConvertedType::TIME_MILLIS;
    col_schema.stats_dtype    = statistics_dtype::dtype_int32;
    col_schema.logical_type   = LogicalType{TimeType{timestamp_is_utc, TimeUnit::MILLIS}};
  }

  template <typename T>
  std::enable_if_t<std::is_same_v<T, cudf::duration_us>, void> operator()()
  {
    col_schema.type           = Type::INT64;
    col_schema.converted_type = ConvertedType::TIME_MICROS;
    col_schema.stats_dtype    = statistics_dtype::dtype_int64;
    col_schema.logical_type   = LogicalType{TimeType{timestamp_is_utc, TimeUnit::MICROS}};
  }

  //  unsupported outside cudf for parquet 1.0.
  template <typename T>
  std::enable_if_t<std::is_same_v<T, cudf::duration_ns>, void> operator()()
  {
    col_schema.type         = Type::INT64;
    col_schema.stats_dtype  = statistics_dtype::dtype_int64;
    col_schema.logical_type = LogicalType{TimeType{timestamp_is_utc, TimeUnit::NANOS}};
  }

  template <typename T>
  std::enable_if_t<cudf::is_fixed_point<T>(), void> operator()()
  {
    if (std::is_same_v<T, numeric::decimal32>) {
      col_schema.type              = Type::INT32;
      col_schema.stats_dtype       = statistics_dtype::dtype_int32;
      col_schema.decimal_precision = MAX_DECIMAL32_PRECISION;
      col_schema.logical_type      = LogicalType{DecimalType{0, MAX_DECIMAL32_PRECISION}};
    } else if (std::is_same_v<T, numeric::decimal64>) {
      col_schema.type              = Type::INT64;
      col_schema.stats_dtype       = statistics_dtype::dtype_decimal64;
      col_schema.decimal_precision = MAX_DECIMAL64_PRECISION;
      col_schema.logical_type      = LogicalType{DecimalType{0, MAX_DECIMAL64_PRECISION}};
    } else if (std::is_same_v<T, numeric::decimal128>) {
      col_schema.type              = Type::FIXED_LEN_BYTE_ARRAY;
      col_schema.type_length       = sizeof(__int128_t);
      col_schema.stats_dtype       = statistics_dtype::dtype_decimal128;
      col_schema.decimal_precision = MAX_DECIMAL128_PRECISION;
      col_schema.logical_type      = LogicalType{DecimalType{0, MAX_DECIMAL128_PRECISION}};
    } else {
      CUDF_FAIL("Unsupported fixed point type for parquet writer");
    }
    col_schema.converted_type = ConvertedType::DECIMAL;
    col_schema.decimal_scale = -col->type().scale();  // parquet and cudf disagree about scale signs
    col_schema.logical_type->decimal_type->scale = -col->type().scale();
    if (col_meta.is_decimal_precision_set()) {
      CUDF_EXPECTS(col_meta.get_decimal_precision() >= col_schema.decimal_scale,
                   "Precision must be equal to or greater than scale!");
      if (col_schema.type == Type::INT64 and col_meta.get_decimal_precision() < 10) {
        CUDF_LOG_WARN("Parquet writer: writing a decimal column with precision < 10 as int64");
      }
      col_schema.decimal_precision                     = col_meta.get_decimal_precision();
      col_schema.logical_type->decimal_type->precision = col_meta.get_decimal_precision();
    }
  }

  template <typename T>
  std::enable_if_t<cudf::is_nested<T>(), void> operator()()
  {
    CUDF_FAIL("This functor is only meant for physical data types");
  }

  template <typename T>
  std::enable_if_t<cudf::is_dictionary<T>(), void> operator()()
  {
    CUDF_FAIL("Dictionary columns are not supported for writing");
  }
};

inline bool is_col_nullable(cudf::detail::LinkedColPtr const& col,
                            column_in_metadata const& col_meta,
                            single_write_mode write_mode)
{
  if (col_meta.is_nullability_defined()) {
    CUDF_EXPECTS(col_meta.nullable() or col->null_count() == 0,
                 "Mismatch in metadata prescribed nullability and input column. "
                 "Metadata for input column with nulls cannot prescribe nullability = false");
    return col_meta.nullable();
  }
  // For chunked write, when not provided nullability, we assume the worst case scenario
  // that all columns are nullable.
  return write_mode == single_write_mode::NO or col->nullable();
}

/**
 * @brief Construct schema from input columns and per-column input options
 *
 * Recursively traverses through linked_columns and corresponding metadata to construct schema tree.
 * The resulting schema tree is stored in a vector in pre-order traversal order.
 */
std::vector<schema_tree_node> construct_schema_tree(
  cudf::detail::LinkedColVector const& linked_columns,
  table_input_metadata& metadata,
  single_write_mode write_mode,
  bool int96_timestamps,
  bool utc_timestamps)
{
  std::vector<schema_tree_node> schema;
  schema_tree_node root{};
  root.type            = UNDEFINED_TYPE;
  root.repetition_type = NO_REPETITION_TYPE;
  root.name            = "schema";
  root.num_children    = linked_columns.size();
  root.parent_idx      = -1;  // root schema has no parent
  schema.push_back(std::move(root));

  std::function<void(cudf::detail::LinkedColPtr const&, column_in_metadata&, size_t)> add_schema =
    [&](cudf::detail::LinkedColPtr const& col, column_in_metadata& col_meta, size_t parent_idx) {
      bool const col_nullable = is_col_nullable(col, col_meta, write_mode);

      auto set_field_id = [&schema, parent_idx](schema_tree_node& s,
                                                column_in_metadata const& col_meta) {
        if (schema[parent_idx].name != "list" and col_meta.is_parquet_field_id_set()) {
          s.field_id = col_meta.get_parquet_field_id();
        }
      };

      auto is_last_list_child = [](cudf::detail::LinkedColPtr col) {
        if (col->type().id() != type_id::LIST) { return false; }
        auto const child_col_type =
          col->children[lists_column_view::child_column_index]->type().id();
        return child_col_type == type_id::UINT8;
      };

      // only call this after col_schema.type has been set
      auto set_encoding = [&schema, parent_idx](schema_tree_node& s,
                                                column_in_metadata const& col_meta) {
        s.requested_encoding = column_encoding::USE_DEFAULT;

        if (s.name != "list" and col_meta.get_encoding() != column_encoding::USE_DEFAULT) {
          // do some validation
          switch (col_meta.get_encoding()) {
            case column_encoding::DELTA_BINARY_PACKED:
              if (s.type != Type::INT32 && s.type != Type::INT64) {
                CUDF_LOG_WARN(
                  "DELTA_BINARY_PACKED encoding is only supported for INT32 and INT64 columns; the "
                  "requested encoding will be ignored");
                return;
              }
              break;

            case column_encoding::DELTA_LENGTH_BYTE_ARRAY:
              if (s.type != Type::BYTE_ARRAY) {
                CUDF_LOG_WARN(
                  "DELTA_LENGTH_BYTE_ARRAY encoding is only supported for BYTE_ARRAY columns; the "
                  "requested encoding will be ignored");
                return;
              }
              // we don't yet allow encoding decimal128 with DELTA_LENGTH_BYTE_ARRAY (nor with
              // the BYTE_ARRAY physical type, but check anyway)
              if (s.converted_type.value_or(ConvertedType::UNKNOWN) == ConvertedType::DECIMAL) {
                CUDF_LOG_WARN(
                  "Decimal types cannot yet be encoded as DELTA_LENGTH_BYTE_ARRAY; the "
                  "requested encoding will be ignored");
                return;
              }
              break;

            case column_encoding::DELTA_BYTE_ARRAY:
              if (s.type != Type::BYTE_ARRAY && s.type != Type::FIXED_LEN_BYTE_ARRAY) {
                CUDF_LOG_WARN(
                  "DELTA_BYTE_ARRAY encoding is only supported for BYTE_ARRAY and "
                  "FIXED_LEN_BYTE_ARRAY columns; the requested encoding will be ignored");
                return;
              }
              // we don't yet allow encoding decimal128 with DELTA_BYTE_ARRAY
              if (s.converted_type.value_or(ConvertedType::UNKNOWN) == ConvertedType::DECIMAL) {
                CUDF_LOG_WARN(
                  "Decimal types cannot yet be encoded as DELTA_BYTE_ARRAY; the "
                  "requested encoding will be ignored");
                return;
              }
              break;

            case column_encoding::BYTE_STREAM_SPLIT:
              if (s.type == Type::BYTE_ARRAY) {
                CUDF_LOG_WARN(
                  "BYTE_STREAM_SPLIT encoding is only supported for fixed width columns; the "
                  "requested encoding will be ignored");
                return;
              }
              if (s.type == Type::INT96) {
                CUDF_LOG_WARN(
                  "BYTE_STREAM_SPLIT encoding is not supported for INT96 columns; the "
                  "requested encoding will be ignored");
                return;
              }
              break;

            // supported parquet encodings
            case column_encoding::PLAIN:
            case column_encoding::DICTIONARY: break;

            // all others
            default:
              CUDF_LOG_WARN(
                "Unsupported page encoding requested: {}; the requested encoding will be ignored",
                static_cast<int>(col_meta.get_encoding()));
              return;
          }

          // requested encoding seems to be ok, set it
          s.requested_encoding = col_meta.get_encoding();
        }
      };

      // There is a special case for a list<int8> column with one byte column child. This column can
      // have a special flag that indicates we write this out as binary instead of a list. This is a
      // more efficient storage mechanism for a single-depth list of bytes, but is a departure from
      // original cuIO behavior so it is locked behind the option. If the option is selected on a
      // column that isn't a single-depth list<int8> the code will throw.
      if (col_meta.is_enabled_output_as_binary() && is_last_list_child(col)) {
        CUDF_EXPECTS(col_meta.num_children() == 2 or col_meta.num_children() == 0,
                     "Binary column's corresponding metadata should have zero or two children");
        if (col_meta.num_children() > 0) {
          CUDF_EXPECTS(col->children[lists_column_view::child_column_index]->children.empty(),
                       "Binary column must not be nested");
        }

        schema_tree_node col_schema{};
        // test if this should be output as FIXED_LEN_BYTE_ARRAY
        if (col_meta.is_type_length_set()) {
          col_schema.type        = Type::FIXED_LEN_BYTE_ARRAY;
          col_schema.type_length = col_meta.get_type_length();
        } else {
          col_schema.type = Type::BYTE_ARRAY;
        }

        col_schema.converted_type  = thrust::nullopt;
        col_schema.stats_dtype     = statistics_dtype::dtype_byte_array;
        col_schema.repetition_type = col_nullable ? OPTIONAL : REQUIRED;
        col_schema.name = (schema[parent_idx].name == "list") ? "element" : col_meta.get_name();
        col_schema.parent_idx  = parent_idx;
        col_schema.leaf_column = col;
        set_field_id(col_schema, col_meta);
        set_encoding(col_schema, col_meta);
        col_schema.output_as_byte_array = col_meta.is_enabled_output_as_binary();
        col_schema.skip_compression     = col_meta.is_enabled_skip_compression();
        schema.push_back(col_schema);
      } else if (col->type().id() == type_id::STRUCT) {
        // if struct, add current and recursively call for all children
        schema_tree_node struct_schema{};
        struct_schema.repetition_type =
          col_nullable ? FieldRepetitionType::OPTIONAL : FieldRepetitionType::REQUIRED;

        struct_schema.name = (schema[parent_idx].name == "list") ? "element" : col_meta.get_name();
        struct_schema.num_children = col->children.size();
        struct_schema.parent_idx   = parent_idx;
        set_field_id(struct_schema, col_meta);
        schema.push_back(std::move(struct_schema));

        auto struct_node_index = schema.size() - 1;
        // for (auto child_it = col->children.begin(); child_it < col->children.end(); child_it++) {
        //   add_schema(*child_it, struct_node_index);
        // }
        CUDF_EXPECTS(col->children.size() == static_cast<size_t>(col_meta.num_children()),
                     "Mismatch in number of child columns between input table and metadata");
        for (size_t i = 0; i < col->children.size(); ++i) {
          add_schema(col->children[i], col_meta.child(i), struct_node_index);
        }
      } else if (col->type().id() == type_id::LIST && !col_meta.is_map()) {
        // List schema is denoted by two levels for each nesting level and one final level for leaf.
        // The top level is the same name as the column name.
        // So e.g. List<List<int>> is denoted in the schema by
        // "col_name" : { "list" : { "element" : { "list" : { "element" } } } }

        schema_tree_node list_schema_1{};
        list_schema_1.converted_type = ConvertedType::LIST;
        list_schema_1.repetition_type =
          col_nullable ? FieldRepetitionType::OPTIONAL : FieldRepetitionType::REQUIRED;
        list_schema_1.name = (schema[parent_idx].name == "list") ? "element" : col_meta.get_name();
        list_schema_1.num_children = 1;
        list_schema_1.parent_idx   = parent_idx;
        set_field_id(list_schema_1, col_meta);
        schema.push_back(std::move(list_schema_1));

        schema_tree_node list_schema_2{};
        list_schema_2.repetition_type = FieldRepetitionType::REPEATED;
        list_schema_2.name            = "list";
        list_schema_2.num_children    = 1;
        list_schema_2.parent_idx      = schema.size() - 1;  // Parent is list_schema_1, last added.
        schema.push_back(std::move(list_schema_2));

        CUDF_EXPECTS(col_meta.num_children() == 2,
                     "List column's metadata should have exactly two children");

        add_schema(col->children[lists_column_view::child_column_index],
                   col_meta.child(lists_column_view::child_column_index),
                   schema.size() - 1);
      } else if (col->type().id() == type_id::LIST && col_meta.is_map()) {
        // Map schema is denoted by a list of struct
        // e.g. List<Struct<String,String>> will be
        // "col_name" : { "key_value" : { "key", "value" } }

        // verify the List child structure is a struct<left_child, right_child>
        column_view struct_col = *col->children[lists_column_view::child_column_index];
        CUDF_EXPECTS(struct_col.type().id() == type_id::STRUCT, "Map should be a List of struct");
        CUDF_EXPECTS(struct_col.num_children() == 2,
                     "Map should be a List of struct with two children only but found " +
                       std::to_string(struct_col.num_children()));

        schema_tree_node map_schema{};
        map_schema.converted_type = ConvertedType::MAP;
        map_schema.repetition_type =
          col_nullable ? FieldRepetitionType::OPTIONAL : FieldRepetitionType::REQUIRED;
        map_schema.name = col_meta.get_name();
        if (col_meta.is_parquet_field_id_set()) {
          map_schema.field_id = col_meta.get_parquet_field_id();
        }
        map_schema.num_children = 1;
        map_schema.parent_idx   = parent_idx;
        schema.push_back(std::move(map_schema));

        schema_tree_node repeat_group{};
        repeat_group.repetition_type = FieldRepetitionType::REPEATED;
        repeat_group.name            = "key_value";
        repeat_group.num_children    = 2;
        repeat_group.parent_idx      = schema.size() - 1;  // Parent is map_schema, last added.
        schema.push_back(std::move(repeat_group));

        CUDF_EXPECTS(col_meta.num_children() == 2,
                     "List column's metadata should have exactly two children");
        CUDF_EXPECTS(col_meta.child(lists_column_view::child_column_index).num_children() == 2,
                     "Map struct column should have exactly two children");
        // verify the col meta of children of the struct have name key and value
        auto& left_child_meta = col_meta.child(lists_column_view::child_column_index).child(0);
        left_child_meta.set_name("key");
        left_child_meta.set_nullability(false);

        auto& right_child_meta = col_meta.child(lists_column_view::child_column_index).child(1);
        right_child_meta.set_name("value");
        // check the repetition type of key is required i.e. the col should be non-nullable
        auto key_col = col->children[lists_column_view::child_column_index]->children[0];
        CUDF_EXPECTS(!is_col_nullable(key_col, left_child_meta, write_mode),
                     "key column cannot be nullable. For chunked writing, explicitly set the "
                     "nullability to false in metadata");
        // process key
        size_type struct_col_index = schema.size() - 1;
        add_schema(key_col, left_child_meta, struct_col_index);
        // process value
        add_schema(col->children[lists_column_view::child_column_index]->children[1],
                   right_child_meta,
                   struct_col_index);

      } else {
        // if leaf, add current
        if (col->type().id() == type_id::STRING) {
          if (col_meta.is_enabled_output_as_binary()) {
            CUDF_EXPECTS(col_meta.num_children() == 2 or col_meta.num_children() == 0,
                         "Binary column's corresponding metadata should have zero or two children");
          } else {
            CUDF_EXPECTS(col_meta.num_children() == 1 or col_meta.num_children() == 0,
                         "String column's corresponding metadata should have zero or one children");
          }
        } else {
          CUDF_EXPECTS(col_meta.num_children() == 0,
                       "Leaf column's corresponding metadata cannot have children");
        }

        schema_tree_node col_schema{};

        bool timestamp_is_int96 = int96_timestamps or col_meta.is_enabled_int96_timestamps();

        cudf::type_dispatcher(
          col->type(),
          leaf_schema_fn{col_schema, col, col_meta, timestamp_is_int96, utc_timestamps});

        col_schema.repetition_type = col_nullable ? OPTIONAL : REQUIRED;
        col_schema.name = (schema[parent_idx].name == "list") ? "element" : col_meta.get_name();
        col_schema.parent_idx  = parent_idx;
        col_schema.leaf_column = col;
        set_field_id(col_schema, col_meta);
        set_encoding(col_schema, col_meta);
        col_schema.skip_compression = col_meta.is_enabled_skip_compression();
        schema.push_back(col_schema);
      }
    };

  CUDF_EXPECTS(metadata.column_metadata.size() == linked_columns.size(),
               "Mismatch in the number of columns and the corresponding metadata elements");
  // Add all linked_columns to schema using parent_idx = 0 (root)
  for (size_t i = 0; i < linked_columns.size(); ++i) {
    add_schema(linked_columns[i], metadata.column_metadata[i], 0);
  }

  return schema;
}

/**
 * @brief Class to store parquet specific information for one data stream.
 *
 * Contains information about a single data stream. In case of struct columns, a data stream is one
 * of the child leaf columns that contains data.
 * e.g. A column Struct<int, List<float>> contains 2 data streams:
 * - Struct<int>
 * - Struct<List<float>>
 *
 */
struct parquet_column_view {
  parquet_column_view(schema_tree_node const& schema_node,
                      std::vector<schema_tree_node> const& schema_tree,
                      rmm::cuda_stream_view stream);

  [[nodiscard]] parquet_column_device_view get_device_view(rmm::cuda_stream_view stream) const;

  [[nodiscard]] column_view cudf_column_view() const { return cudf_col; }
  [[nodiscard]] Type physical_type() const { return schema_node.type; }
  [[nodiscard]] ConvertedType converted_type() const
  {
    return schema_node.converted_type.value_or(UNKNOWN);
  }

  std::vector<std::string> const& get_path_in_schema() { return path_in_schema; }

  // LIST related member functions
  [[nodiscard]] uint8_t max_def_level() const noexcept { return _max_def_level; }
  [[nodiscard]] uint8_t max_rep_level() const noexcept { return _max_rep_level; }
  [[nodiscard]] bool is_list() const noexcept { return _is_list; }

 private:
  // Schema related members
  schema_tree_node schema_node;
  std::vector<std::string> path_in_schema;
  uint8_t _max_def_level = 0;
  uint8_t _max_rep_level = 0;
  rmm::device_uvector<uint8_t> _d_nullability;

  column_view cudf_col;

  // List-related members
  bool _is_list;
  rmm::device_uvector<size_type>
    _dremel_offsets;  ///< For each row, the absolute offset into the repetition and definition
                      ///< level vectors. O(num rows)
  rmm::device_uvector<uint8_t> _rep_level;
  rmm::device_uvector<uint8_t> _def_level;
  std::vector<uint8_t> _nullability;
  size_type _data_count = 0;
};

parquet_column_view::parquet_column_view(schema_tree_node const& schema_node,
                                         std::vector<schema_tree_node> const& schema_tree,
                                         rmm::cuda_stream_view stream)
  : schema_node(schema_node),
    _d_nullability(0, stream),
    _dremel_offsets(0, stream),
    _rep_level(0, stream),
    _def_level(0, stream)
{
  // Construct single inheritance column_view from linked_column_view
  auto curr_col                           = schema_node.leaf_column.get();
  column_view single_inheritance_cudf_col = *curr_col;
  while (curr_col->parent) {
    auto const& parent = *curr_col->parent;

    // For list columns, we still need to retain the offset child column.
    auto children =
      (parent.type().id() == type_id::LIST)
        ? std::vector<column_view>{*parent.children[lists_column_view::offsets_column_index],
                                   single_inheritance_cudf_col}
        : std::vector<column_view>{single_inheritance_cudf_col};

    single_inheritance_cudf_col = column_view(parent.type(),
                                              parent.size(),
                                              parent.head(),
                                              parent.null_mask(),
                                              parent.null_count(),
                                              parent.offset(),
                                              children);

    curr_col = curr_col->parent;
  }
  cudf_col = single_inheritance_cudf_col;

  // Construct path_in_schema by travelling up in the schema_tree
  std::vector<std::string> path;
  auto curr_schema_node = schema_node;
  do {
    path.push_back(curr_schema_node.name);
    if (curr_schema_node.parent_idx != -1) {
      curr_schema_node = schema_tree[curr_schema_node.parent_idx];
    }
  } while (curr_schema_node.parent_idx != -1);
  path_in_schema = std::vector<std::string>(path.crbegin(), path.crend());

  // Calculate max definition level by counting the number of levels that are optional (nullable)
  // and max repetition level by counting the number of REPEATED levels in this column's hierarchy
  uint16_t max_def_level = 0;
  uint16_t max_rep_level = 0;
  curr_schema_node       = schema_node;
  while (curr_schema_node.parent_idx != -1) {
    if (curr_schema_node.repetition_type == REPEATED or
        curr_schema_node.repetition_type == OPTIONAL) {
      ++max_def_level;
    }
    if (curr_schema_node.repetition_type == REPEATED) { ++max_rep_level; }
    curr_schema_node = schema_tree[curr_schema_node.parent_idx];
  }
  CUDF_EXPECTS(max_def_level < 256, "Definition levels above 255 are not supported");
  CUDF_EXPECTS(max_rep_level < 256, "Definition levels above 255 are not supported");

  _max_def_level = max_def_level;
  _max_rep_level = max_rep_level;

  // Construct nullability vector using repetition_type from schema.
  std::vector<uint8_t> r_nullability;
  curr_schema_node = schema_node;
  while (curr_schema_node.parent_idx != -1) {
    if (not curr_schema_node.is_stub()) {
      r_nullability.push_back(curr_schema_node.repetition_type == FieldRepetitionType::OPTIONAL);
    }
    curr_schema_node = schema_tree[curr_schema_node.parent_idx];
  }
  _nullability = std::vector<uint8_t>(r_nullability.crbegin(), r_nullability.crend());
  // TODO(cp): Explore doing this for all columns in a single go outside this ctor. Maybe using
  // hostdevice_vector. Currently this involves a hipMemcpyAsync for each column.
  _d_nullability = cudf::detail::make_device_uvector_async(
    _nullability, stream, rmm::mr::get_current_device_resource());

  _is_list = (_max_rep_level > 0);

  if (cudf_col.is_empty()) { return; }

  if (_is_list) {
    // Top level column's offsets are not applied to all children. Get the effective offset and
    // size of the leaf column
    // Calculate row offset into dremel data (repetition/definition values) and the respective
    // definition and repetition levels
    cudf::detail::dremel_data dremel =
      get_dremel_data(cudf_col, _nullability, schema_node.output_as_byte_array, stream);
    _dremel_offsets = std::move(dremel.dremel_offsets);
    _rep_level      = std::move(dremel.rep_level);
    _def_level      = std::move(dremel.def_level);
    _data_count     = dremel.leaf_data_size;  // Needed for knowing what size dictionary to allocate

    stream.synchronize();
  } else {
    // For non-list struct, the size of the root column is the same as the size of the leaf column
    _data_count = cudf_col.size();
  }
}

parquet_column_device_view parquet_column_view::get_device_view(rmm::cuda_stream_view) const
{
  auto desc        = parquet_column_device_view{};  // Zero out all fields
  desc.stats_dtype = schema_node.stats_dtype;
  desc.ts_scale    = schema_node.ts_scale;
  desc.type_length = schema_node.type_length;

  if (is_list()) {
    desc.level_offsets = _dremel_offsets.data();
    desc.rep_values    = _rep_level.data();
    desc.def_values    = _def_level.data();
  }
  desc.num_rows             = cudf_col.size();
  desc.physical_type        = physical_type();
  desc.converted_type       = converted_type();
  desc.output_as_byte_array = schema_node.output_as_byte_array;

  desc.level_bits = CompactProtocolReader::NumRequiredBits(max_rep_level()) << 4 |
                    CompactProtocolReader::NumRequiredBits(max_def_level());
  desc.nullability        = _d_nullability.data();
  desc.max_def_level      = _max_def_level;
  desc.max_rep_level      = _max_rep_level;
  desc.requested_encoding = schema_node.requested_encoding;
  desc.skip_compression   = schema_node.skip_compression;
  return desc;
}

/**
 * @brief Gather row group fragments
 *
 * This calculates fragments to be used in determining row group boundaries.
 *
 * @param frag Destination row group fragments
 * @param col_desc column description array
 * @param partitions Information about partitioning of table
 * @param part_frag_offset A Partition's offset into fragment array
 * @param fragment_size Number of rows per fragment
 * @param stream CUDA stream used for device memory operations and kernel launches
 */
void init_row_group_fragments(cudf::detail::hostdevice_2dvector<PageFragment>& frag,
                              device_span<parquet_column_device_view const> col_desc,
                              host_span<partition_info const> partitions,
                              device_span<int const> part_frag_offset,
                              uint32_t fragment_size,
                              rmm::cuda_stream_view stream)
{
  auto d_partitions = cudf::detail::make_device_uvector_async(
    partitions, stream, rmm::mr::get_current_device_resource());
  InitRowGroupFragments(frag, col_desc, d_partitions, part_frag_offset, fragment_size, stream);
  frag.device_to_host_sync(stream);
}

/**
 * @brief Recalculate page fragments
 *
 * This calculates fragments to be used to determine page boundaries within
 * column chunks.
 *
 * @param frag Destination page fragments
 * @param frag_sizes Array of fragment sizes for each column
 * @param stream CUDA stream used for device memory operations and kernel launches
 */
void calculate_page_fragments(device_span<PageFragment> frag,
                              host_span<size_type const> frag_sizes,
                              rmm::cuda_stream_view stream)
{
  auto d_frag_sz = cudf::detail::make_device_uvector_async(
    frag_sizes, stream, rmm::mr::get_current_device_resource());
  CalculatePageFragments(frag, d_frag_sz, stream);
}

/**
 * @brief Gather per-fragment statistics
 *
 * @param frag_stats output statistics
 * @param frags Input page fragments
 * @param int96_timestamps Flag to indicate if timestamps will be written as INT96
 * @param stream CUDA stream used for device memory operations and kernel launches
 */
void gather_fragment_statistics(device_span<statistics_chunk> frag_stats,
                                device_span<PageFragment const> frags,
                                bool int96_timestamps,
                                rmm::cuda_stream_view stream)
{
  rmm::device_uvector<statistics_group> frag_stats_group(frag_stats.size(), stream);

  InitFragmentStatistics(frag_stats_group, frags, stream);
  detail::calculate_group_statistics<detail::io_file_format::PARQUET>(
    frag_stats.data(), frag_stats_group.data(), frag_stats.size(), stream, int96_timestamps);
  stream.synchronize();
}

auto to_nvcomp_compression_type(Compression codec)
{
  if (codec == Compression::SNAPPY) return nvcomp::compression_type::SNAPPY;
  if (codec == Compression::ZSTD) return nvcomp::compression_type::ZSTD;
  // Parquet refers to LZ4 as "LZ4_RAW"; Parquet's "LZ4" is not standard LZ4
  if (codec == Compression::LZ4_RAW) return nvcomp::compression_type::LZ4;
  CUDF_FAIL("Unsupported compression type");
}

auto page_alignment(Compression codec)
{
  if (codec == Compression::UNCOMPRESSED or
      nvcomp::is_compression_disabled(to_nvcomp_compression_type(codec))) {
    return 1u;
  }

  return 1u << nvcomp::compress_input_alignment_bits(to_nvcomp_compression_type(codec));
}

size_t max_compression_output_size(Compression codec, uint32_t compression_blocksize)
{
  if (codec == Compression::UNCOMPRESSED) return 0;

  return compress_max_output_chunk_size(to_nvcomp_compression_type(codec), compression_blocksize);
}

auto init_page_sizes(hostdevice_2dvector<EncColumnChunk>& chunks,
                     device_span<parquet_column_device_view const> col_desc,
                     uint32_t num_columns,
                     size_t max_page_size_bytes,
                     size_type max_page_size_rows,
                     bool write_v2_headers,
                     Compression compression_codec,
                     rmm::cuda_stream_view stream)
{
  if (chunks.is_empty()) { return cudf::detail::hostdevice_vector<size_type>{}; }

  chunks.host_to_device_async(stream);
  // Calculate number of pages and store in respective chunks
  InitEncoderPages(chunks,
                   {},
                   {},
                   {},
                   col_desc,
                   num_columns,
                   max_page_size_bytes,
                   max_page_size_rows,
                   page_alignment(compression_codec),
                   write_v2_headers,
                   nullptr,
                   nullptr,
                   stream);
  chunks.device_to_host_sync(stream);

  int num_pages = 0;
  for (auto& chunk : chunks.host_view().flat_view()) {
    chunk.first_page = num_pages;
    num_pages += chunk.num_pages;
  }
  chunks.host_to_device_async(stream);

  // Now that we know the number of pages, allocate an array to hold per page size and get it
  // populated
  cudf::detail::hostdevice_vector<size_type> page_sizes(num_pages, stream);
  InitEncoderPages(chunks,
                   {},
                   page_sizes,
                   {},
                   col_desc,
                   num_columns,
                   max_page_size_bytes,
                   max_page_size_rows,
                   page_alignment(compression_codec),
                   write_v2_headers,
                   nullptr,
                   nullptr,
                   stream);
  page_sizes.device_to_host_sync(stream);

  // Get per-page max compressed size
  cudf::detail::hostdevice_vector<size_type> comp_page_sizes(num_pages, stream);
  std::transform(page_sizes.begin(),
                 page_sizes.end(),
                 comp_page_sizes.begin(),
                 [compression_codec](auto page_size) {
                   return max_compression_output_size(compression_codec, page_size);
                 });
  comp_page_sizes.host_to_device_async(stream);

  // Use per-page max compressed size to calculate chunk.compressed_size
  InitEncoderPages(chunks,
                   {},
                   {},
                   comp_page_sizes,
                   col_desc,
                   num_columns,
                   max_page_size_bytes,
                   max_page_size_rows,
                   page_alignment(compression_codec),
                   write_v2_headers,
                   nullptr,
                   nullptr,
                   stream);
  chunks.device_to_host_sync(stream);
  return comp_page_sizes;
}

size_t max_page_bytes(Compression compression, size_t max_page_size_bytes)
{
  if (compression == Compression::UNCOMPRESSED) { return max_page_size_bytes; }

  auto const ncomp_type   = to_nvcomp_compression_type(compression);
  auto const nvcomp_limit = nvcomp::is_compression_disabled(ncomp_type)
                              ? std::nullopt
                              : nvcomp::compress_max_allowed_chunk_size(ncomp_type);

  auto max_size = std::min(nvcomp_limit.value_or(max_page_size_bytes), max_page_size_bytes);
  // page size must fit in a 32-bit signed integer
  return std::min<size_t>(max_size, std::numeric_limits<int32_t>::max());
}

std::pair<std::vector<rmm::device_uvector<size_type>>, std::vector<rmm::device_uvector<size_type>>>
build_chunk_dictionaries(hostdevice_2dvector<EncColumnChunk>& chunks,
                         host_span<parquet_column_device_view const> col_desc,
                         device_2dspan<PageFragment const> frags,
                         Compression compression,
                         dictionary_policy dict_policy,
                         size_t max_dict_size,
                         rmm::cuda_stream_view stream)
{
  // At this point, we know all chunks and their sizes. We want to allocate dictionaries for each
  // chunk that can have dictionary

  auto h_chunks = chunks.host_view().flat_view();

  std::vector<rmm::device_uvector<size_type>> dict_data;
  std::vector<rmm::device_uvector<size_type>> dict_index;

  if (h_chunks.empty()) { return std::pair(std::move(dict_data), std::move(dict_index)); }

  if (dict_policy == dictionary_policy::NEVER) {
    thrust::for_each(
      h_chunks.begin(), h_chunks.end(), [](auto& chunk) { chunk.use_dictionary = false; });
    chunks.host_to_device_async(stream);
    return std::pair(std::move(dict_data), std::move(dict_index));
  }

  // Allocate slots for each chunk
  std::vector<rmm::device_uvector<slot_type>> hash_maps_storage;
  hash_maps_storage.reserve(h_chunks.size());
  for (auto& chunk : h_chunks) {
    auto const& chunk_col_desc = col_desc[chunk.col_desc_id];
    auto const is_requested_non_dict =
      chunk_col_desc.requested_encoding != column_encoding::USE_DEFAULT &&
      chunk_col_desc.requested_encoding != column_encoding::DICTIONARY;
    auto const is_type_non_dict =
      chunk_col_desc.physical_type == Type::BOOLEAN || chunk_col_desc.output_as_byte_array;

    if (is_type_non_dict || is_requested_non_dict) {
      chunk.use_dictionary = false;
    } else {
      chunk.use_dictionary = true;
      // cuCollections suggests using a hash map of size N * (1/0.7) = num_values * 1.43
      // https://github.com/NVIDIA/cuCollections/blob/3a49fc71/include/cuco/static_map.cuh#L190-L193
      auto& inserted_map   = hash_maps_storage.emplace_back(chunk.num_values * 1.43, stream);
      chunk.dict_map_slots = inserted_map.data();
      chunk.dict_map_size  = inserted_map.size();
    }
  }

  chunks.host_to_device_async(stream);

  initialize_chunk_hash_maps(chunks.device_view().flat_view(), stream);
  populate_chunk_hash_maps(frags, stream);

  chunks.device_to_host_sync(stream);

  // Make decision about which chunks have dictionary
  bool cannot_honor_request = false;
  for (auto& ck : h_chunks) {
    if (not ck.use_dictionary) { continue; }
    std::tie(ck.use_dictionary, ck.dict_rle_bits) = [&]() -> std::pair<bool, uint8_t> {
      // calculate size of chunk if dictionary is used

      // If we have N unique values then the idx for the last value is N - 1 and nbits is the number
      // of bits required to encode indices into the dictionary
      auto max_dict_index = (ck.num_dict_entries > 0) ? ck.num_dict_entries - 1 : 0;
      auto nbits          = std::max(CompactProtocolReader::NumRequiredBits(max_dict_index), 1);

      // We don't use dictionary if the indices are > MAX_DICT_BITS bits because that's the maximum
      // bitpacking bitsize we efficiently support
      if (nbits > MAX_DICT_BITS) { return {false, 0}; }

      auto rle_byte_size = util::div_rounding_up_safe(ck.num_values * nbits, 8);
      auto dict_enc_size = ck.uniq_data_size + rle_byte_size;
      if (ck.plain_data_size <= dict_enc_size) { return {false, 0}; }

      // don't use dictionary if it gets too large for the given compression codec
      if (dict_policy == dictionary_policy::ADAPTIVE) {
        auto const unique_size = static_cast<size_t>(ck.uniq_data_size);
        if (unique_size > max_page_bytes(compression, max_dict_size)) { return {false, 0}; }
      }

      return {true, nbits};
    }();
    // If dictionary encoding was requested, but it cannot be used, then print a warning. It will
    // actually be disabled in gpuInitPages.
    if (not ck.use_dictionary) {
      auto const& chunk_col_desc = col_desc[ck.col_desc_id];
      if (chunk_col_desc.requested_encoding == column_encoding::DICTIONARY) {
        cannot_honor_request = true;
      }
    }
  }

  // warn if we have to ignore requested encoding
  if (cannot_honor_request) {
    CUDF_LOG_WARN("DICTIONARY encoding was requested, but resource constraints prevent its use");
  }

  // TODO: (enh) Deallocate hash map storage for chunks that don't use dict and clear pointers.

  dict_data.reserve(h_chunks.size());
  dict_index.reserve(h_chunks.size());
  for (auto& chunk : h_chunks) {
    if (not chunk.use_dictionary) { continue; }

    size_t dict_data_size     = std::min(MAX_DICT_SIZE, chunk.dict_map_size);
    auto& inserted_dict_data  = dict_data.emplace_back(dict_data_size, stream);
    auto& inserted_dict_index = dict_index.emplace_back(chunk.num_values, stream);
    chunk.dict_data           = inserted_dict_data.data();
    chunk.dict_index          = inserted_dict_index.data();
  }
  chunks.host_to_device_async(stream);
  collect_map_entries(chunks.device_view().flat_view(), stream);
  get_dictionary_indices(frags, stream);

  return std::pair(std::move(dict_data), std::move(dict_index));
}

/**
 * @brief Initialize encoder pages.
 *
 * @param chunks Column chunk array
 * @param col_desc Column description array
 * @param pages Encoder pages array
 * @param comp_page_sizes Per-page max compressed size
 * @param page_stats Page statistics array
 * @param frag_stats Fragment statistics array
 * @param num_columns Total number of columns
 * @param num_pages Total number of pages
 * @param num_stats_bfr Number of statistics buffers
 * @param compression Compression format
 * @param max_page_size_bytes Maximum uncompressed page size, in bytes
 * @param max_page_size_rows Maximum page size, in rows
 * @param write_v2_headers True if version 2 page headers are to be written
 * @param stream CUDA stream used for device memory operations and kernel launches
 */
void init_encoder_pages(hostdevice_2dvector<EncColumnChunk>& chunks,
                        device_span<parquet_column_device_view const> col_desc,
                        device_span<EncPage> pages,
                        cudf::detail::hostdevice_vector<size_type>& comp_page_sizes,
                        statistics_chunk* page_stats,
                        statistics_chunk* frag_stats,
                        uint32_t num_columns,
                        uint32_t num_pages,
                        uint32_t num_stats_bfr,
                        Compression compression,
                        size_t max_page_size_bytes,
                        size_type max_page_size_rows,
                        bool write_v2_headers,
                        rmm::cuda_stream_view stream)
{
  rmm::device_uvector<statistics_merge_group> page_stats_mrg(num_stats_bfr, stream);
  chunks.host_to_device_async(stream);
  InitEncoderPages(chunks,
                   pages,
                   {},
                   comp_page_sizes,
                   col_desc,
                   num_columns,
                   max_page_size_bytes,
                   max_page_size_rows,
                   page_alignment(compression),
                   write_v2_headers,
                   (num_stats_bfr) ? page_stats_mrg.data() : nullptr,
                   (num_stats_bfr > num_pages) ? page_stats_mrg.data() + num_pages : nullptr,
                   stream);
  if (num_stats_bfr > 0) {
    detail::merge_group_statistics<detail::io_file_format::PARQUET>(
      page_stats, frag_stats, page_stats_mrg.data(), num_pages, stream);
    if (num_stats_bfr > num_pages) {
      detail::merge_group_statistics<detail::io_file_format::PARQUET>(
        page_stats + num_pages,
        page_stats,
        page_stats_mrg.data() + num_pages,
        num_stats_bfr - num_pages,
        stream);
    }
  }
  stream.synchronize();
}

/**
 * @brief Encode pages.
 *
 * @throws rmm::bad_alloc if there is insufficient space for temporary buffers
 *
 * @param chunks column chunk array
 * @param pages encoder pages array
 * @param num_rowgroups number of rowgroups
 * @param page_stats optional page-level statistics (nullptr if none)
 * @param chunk_stats optional chunk-level statistics (nullptr if none)
 * @param column_stats optional page-level statistics for column index (nullptr if none)
 * @param comp_stats optional compression statistics (nullopt if none)
 * @param compression compression format
 * @param column_index_truncate_length maximum length of min or max values in column index, in bytes
 * @param write_v2_headers True if V2 page headers should be written
 * @param stream CUDA stream used for device memory operations and kernel launches
 */
void encode_pages(hostdevice_2dvector<EncColumnChunk>& chunks,
                  device_span<EncPage> pages,
                  statistics_chunk const* page_stats,
                  statistics_chunk const* chunk_stats,
                  statistics_chunk const* column_stats,
                  std::optional<writer_compression_statistics>& comp_stats,
                  Compression compression,
                  int32_t column_index_truncate_length,
                  bool write_v2_headers,
                  rmm::cuda_stream_view stream)
{
  auto const num_pages = pages.size();
  auto pages_stats     = (page_stats != nullptr)
                           ? device_span<statistics_chunk const>(page_stats, num_pages)
                           : device_span<statistics_chunk const>();

  uint32_t max_comp_pages = (compression != Compression::UNCOMPRESSED) ? num_pages : 0;

  rmm::device_uvector<device_span<uint8_t const>> comp_in(max_comp_pages, stream);
  rmm::device_uvector<device_span<uint8_t>> comp_out(max_comp_pages, stream);
  rmm::device_uvector<compression_result> comp_res(max_comp_pages, stream);
  thrust::fill(rmm::exec_policy(stream),
               comp_res.begin(),
               comp_res.end(),
               compression_result{0, compression_status::FAILURE});

  EncodePages(pages, write_v2_headers, comp_in, comp_out, comp_res, stream);
  switch (compression) {
    case Compression::SNAPPY:
      if (nvcomp::is_compression_disabled(nvcomp::compression_type::SNAPPY)) {
        gpu_snap(comp_in, comp_out, comp_res, stream);
      } else {
        nvcomp::batched_compress(
          nvcomp::compression_type::SNAPPY, comp_in, comp_out, comp_res, stream);
      }
      break;
    case Compression::ZSTD: {
      if (auto const reason = nvcomp::is_compression_disabled(nvcomp::compression_type::ZSTD);
          reason) {
        CUDF_FAIL("Compression error: " + reason.value());
      }
      nvcomp::batched_compress(nvcomp::compression_type::ZSTD, comp_in, comp_out, comp_res, stream);
      break;
    }
    case Compression::LZ4_RAW: {
      if (auto const reason = nvcomp::is_compression_disabled(nvcomp::compression_type::LZ4);
          reason) {
        CUDF_FAIL("Compression error: " + reason.value());
      }
      nvcomp::batched_compress(nvcomp::compression_type::LZ4, comp_in, comp_out, comp_res, stream);
      break;
    }
    case Compression::UNCOMPRESSED: break;
    default: CUDF_FAIL("invalid compression type");
  }

  // TBD: Not clear if the official spec actually allows dynamically turning off compression at the
  // chunk-level

  auto d_chunks = chunks.device_view();
  DecideCompression(d_chunks.flat_view(), stream);
  EncodePageHeaders(pages, comp_res, pages_stats, chunk_stats, stream);
  GatherPages(d_chunks.flat_view(), pages, stream);

  // By now, the var_bytes has been calculated in InitPages, and the histograms in EncodePages.
  // EncodeColumnIndexes can encode the histograms in the ColumnIndex, and also sum up var_bytes
  // and the histograms for inclusion in the chunk's SizeStats.
  if (column_stats != nullptr) {
    EncodeColumnIndexes(
      d_chunks.flat_view(), {column_stats, pages.size()}, column_index_truncate_length, stream);
  }

  auto h_chunks = chunks.host_view();
  CUDF_CUDA_TRY(hipMemcpyAsync(h_chunks.data(),
                                d_chunks.data(),
                                d_chunks.flat_view().size_bytes(),
                                hipMemcpyDefault,
                                stream.value()));

  if (comp_stats.has_value()) {
    comp_stats.value() += collect_compression_statistics(comp_in, comp_res, stream);
  }
  stream.synchronize();
}

/**
 * @brief Function to calculate the memory needed to encode the column index of the given
 * column chunk.
 *
 * @param ck pointer to column chunk
 * @param col `parquet_column_device_view` for the column
 * @param column_index_truncate_length maximum length of min or max values in column index, in bytes
 * @return Computed buffer size needed to encode the column index
 */
size_t column_index_buffer_size(EncColumnChunk* ck,
                                parquet_column_device_view const& col,
                                int32_t column_index_truncate_length)
{
  // encoding the column index for a given chunk requires:
  //   each list (4 of them) requires 6 bytes of overhead
  //     (1 byte field header, 1 byte type, 4 bytes length)
  //   1 byte overhead for boundary_order
  //   1 byte overhead for termination
  //   sizeof(char) for boundary_order
  //   sizeof(bool) * num_pages for null_pages
  //   (ck_max_stats_len + 4) * num_pages * 2 for min/max values
  //     (each binary requires 4 bytes length + ck_max_stats_len)
  //   sizeof(int64_t) * num_pages for null_counts
  //
  // so 26 bytes overhead + sizeof(char) +
  //    (sizeof(bool) + sizeof(int64_t) + 2 * (4 + ck_max_stats_len)) * num_pages
  //
  // we already have ck->ck_stat_size = 48 + 2 * ck_max_stats_len
  // all of the overhead and non-stats data can fit in under 48 bytes
  //
  // so we can simply use ck_stat_size * num_pages
  //
  // add on some extra padding at the end (plus extra 7 bytes of alignment padding)
  // for scratch space to do stats truncation.

  // additional storage needed for SizeStatistics
  // don't need stats for dictionary pages
  auto const num_pages = ck->num_data_pages();

  // only need variable length size info for BYTE_ARRAY
  // 1 byte for marker, 1 byte vec type, 4 bytes length, 5 bytes per page for values
  // (5 bytes is needed because the varint encoder only encodes 7 bits per byte)
  auto const var_bytes_size = col.physical_type == BYTE_ARRAY ? 6 + 5 * num_pages : 0;

  // for the histograms, need 1 byte for marker, 1 byte vec type, 4 bytes length,
  // (max_level + 1) * 5 bytes per page
  auto const has_def       = col.max_def_level > DEF_LVL_HIST_CUTOFF;
  auto const has_rep       = col.max_def_level > REP_LVL_HIST_CUTOFF;
  auto const def_hist_size = has_def ? 6 + 5 * num_pages * (col.max_def_level + 1) : 0;
  auto const rep_hist_size = has_rep ? 6 + 5 * num_pages * (col.max_rep_level + 1) : 0;

  // total size of SizeStruct is 1 byte marker, 1 byte end-of-struct, plus sizes for components
  auto const size_struct_size = 2 + def_hist_size + rep_hist_size + var_bytes_size;

  // calculating this per-chunk because the sizes can be wildly different.
  constexpr size_t padding = 7;
  return ck->ck_stat_size * num_pages + column_index_truncate_length + padding + size_struct_size;
}

/**
 * @brief Fill the table metadata with default column names.
 *
 * @param table_meta The table metadata to fill
 */
void fill_table_meta(std::unique_ptr<table_input_metadata> const& table_meta)
{
  // Fill unnamed columns' names in table_meta
  std::function<void(column_in_metadata&, std::string)> add_default_name =
    [&](column_in_metadata& col_meta, std::string default_name) {
      if (col_meta.get_name().empty()) col_meta.set_name(default_name);
      for (size_type i = 0; i < col_meta.num_children(); ++i) {
        add_default_name(col_meta.child(i), col_meta.get_name() + "_" + std::to_string(i));
      }
    };
  for (size_t i = 0; i < table_meta->column_metadata.size(); ++i) {
    add_default_name(table_meta->column_metadata[i], "_col" + std::to_string(i));
  }
}

/**
 * @brief Perform the processing steps needed to convert the input table into the output Parquet
 * data for writing, such as compression and encoding.
 *
 * @param[in,out] table_meta The table metadata
 * @param input The input table
 * @param partitions Optional partitions to divide the table into, if specified then must be same
 *        size as number of sinks
 * @param kv_meta Optional user metadata
 * @param curr_agg_meta The current aggregate writer metadata
 * @param max_page_fragment_size_opt Optional maximum number of rows in a page fragment
 * @param max_row_group_size Maximum row group size, in bytes
 * @param max_page_size_bytes Maximum uncompressed page size, in bytes
 * @param max_row_group_rows Maximum row group size, in rows
 * @param max_page_size_rows Maximum page size, in rows
 * @param column_index_truncate_length maximum length of min or max values in column index, in bytes
 * @param stats_granularity Level of statistics requested in output file
 * @param compression Compression format
 * @param collect_statistics Flag to indicate if statistics should be collected
 * @param dict_policy Policy for dictionary use
 * @param max_dictionary_size Maximum dictionary size, in bytes
 * @param single_write_mode Flag to indicate that we are guaranteeing a single table write
 * @param int96_timestamps Flag to indicate if timestamps will be written as INT96
 * @param utc_timestamps Flag to indicate if timestamps are UTC
 * @param write_v2_headers True if V2 page headers are to be written
 * @param out_sink Sink for checking if device write is supported, should not be used to write any
 *        data in this function
 * @param stream CUDA stream used for device memory operations and kernel launches
 * @return A tuple of the intermediate results containing the processed data
 */
auto convert_table_to_parquet_data(table_input_metadata& table_meta,
                                   table_view const& input,
                                   host_span<partition_info const> partitions,
                                   host_span<std::map<std::string, std::string> const> kv_meta,
                                   std::unique_ptr<aggregate_writer_metadata> const& curr_agg_meta,
                                   std::optional<size_type> max_page_fragment_size_opt,
                                   size_t max_row_group_size,
                                   size_t max_page_size_bytes,
                                   size_type max_row_group_rows,
                                   size_type max_page_size_rows,
                                   int32_t column_index_truncate_length,
                                   statistics_freq stats_granularity,
                                   Compression compression,
                                   bool collect_compression_statistics,
                                   dictionary_policy dict_policy,
                                   size_t max_dictionary_size,
                                   single_write_mode write_mode,
                                   bool int96_timestamps,
                                   bool utc_timestamps,
                                   bool write_v2_headers,
                                   host_span<std::unique_ptr<data_sink> const> out_sink,
                                   rmm::cuda_stream_view stream)
{
  auto vec = table_to_linked_columns(input);
  auto schema_tree =
    construct_schema_tree(vec, table_meta, write_mode, int96_timestamps, utc_timestamps);
  // Construct parquet_column_views from the schema tree leaf nodes.
  std::vector<parquet_column_view> parquet_columns;

  for (schema_tree_node const& schema_node : schema_tree) {
    if (schema_node.leaf_column) { parquet_columns.emplace_back(schema_node, schema_tree, stream); }
  }

  // Mass allocation of column_device_views for each parquet_column_view
  std::vector<column_view> cudf_cols;
  cudf_cols.reserve(parquet_columns.size());
  for (auto const& parq_col : parquet_columns) {
    cudf_cols.push_back(parq_col.cudf_column_view());
  }
  table_view single_streams_table(cudf_cols);
  size_type num_columns = single_streams_table.num_columns();

  std::vector<SchemaElement> this_table_schema(schema_tree.begin(), schema_tree.end());

  // Initialize column description
  cudf::detail::hostdevice_vector<parquet_column_device_view> col_desc(parquet_columns.size(),
                                                                       stream);
  std::transform(
    parquet_columns.begin(), parquet_columns.end(), col_desc.host_ptr(), [&](auto const& pcol) {
      return pcol.get_device_view(stream);
    });

  // Init page fragments
  // 5000 is good enough for up to ~200-character strings. Longer strings and deeply nested columns
  // will start producing fragments larger than the desired page size, so calculate fragment sizes
  // for each leaf column.  Skip if the fragment size is not the default.
  size_type max_page_fragment_size =
    max_page_fragment_size_opt.value_or(default_max_page_fragment_size);

  std::vector<size_type> column_frag_size(num_columns, max_page_fragment_size);

  if (input.num_rows() > 0 && not max_page_fragment_size_opt.has_value()) {
    std::vector<size_t> column_sizes;
    std::transform(single_streams_table.begin(),
                   single_streams_table.end(),
                   std::back_inserter(column_sizes),
                   [&](auto const& column) { return column_size(column, stream); });

    // adjust global fragment size if a single fragment will overrun a rowgroup
    auto const table_size  = std::reduce(column_sizes.begin(), column_sizes.end());
    auto const avg_row_len = util::div_rounding_up_safe<size_t>(table_size, input.num_rows());
    if (avg_row_len > 0) {
      auto const rg_frag_size = util::div_rounding_up_safe(max_row_group_size, avg_row_len);
      max_page_fragment_size  = std::min<size_type>(rg_frag_size, max_page_fragment_size);
    }

    // dividing page size by average row length will tend to overshoot the desired
    // page size when there's high variability in the row lengths. instead, shoot
    // for multiple fragments per page to smooth things out. using 2 was too
    // unbalanced in final page sizes, so using 4 which seems to be a good
    // compromise at smoothing things out without getting fragment sizes too small.
    auto frag_size_fn = [&](auto const& col, size_type col_size) {
      int const target_frags_per_page = is_col_fixed_width(col) ? 1 : 4;
      auto const avg_len =
        target_frags_per_page * util::div_rounding_up_safe<size_type>(col_size, input.num_rows());
      if (avg_len > 0) {
        auto const frag_size = util::div_rounding_up_safe<size_type>(max_page_size_bytes, avg_len);
        return std::min<size_type>(max_page_fragment_size, frag_size);
      } else {
        return max_page_fragment_size;
      }
    };

    std::transform(single_streams_table.begin(),
                   single_streams_table.end(),
                   column_sizes.begin(),
                   column_frag_size.begin(),
                   frag_size_fn);
  }

  // Fragments are calculated in two passes. In the first pass, a uniform number of fragments
  // per column is used. This is done to satisfy the requirement that each column chunk within
  // a row group has the same number of rows. After the row group (and thus column chunk)
  // boundaries are known, a second pass is done to calculate fragments to be used in determining
  // page boundaries within each column chunk.
  std::vector<int> num_frag_in_part;
  std::transform(partitions.begin(),
                 partitions.end(),
                 std::back_inserter(num_frag_in_part),
                 [max_page_fragment_size](auto const& part) {
                   return util::div_rounding_up_unsafe(part.num_rows, max_page_fragment_size);
                 });

  size_type num_fragments = std::reduce(num_frag_in_part.begin(), num_frag_in_part.end());

  std::vector<int> part_frag_offset;  // Store the idx of the first fragment in each partition
  std::exclusive_scan(
    num_frag_in_part.begin(), num_frag_in_part.end(), std::back_inserter(part_frag_offset), 0);
  part_frag_offset.push_back(part_frag_offset.back() + num_frag_in_part.back());

  auto d_part_frag_offset = cudf::detail::make_device_uvector_async(
    part_frag_offset, stream, rmm::mr::get_current_device_resource());
  cudf::detail::hostdevice_2dvector<PageFragment> row_group_fragments(
    num_columns, num_fragments, stream);

  // Create table_device_view so that corresponding column_device_view data
  // can be written into col_desc members
  // These are unused but needs to be kept alive.
  auto parent_column_table_device_view = table_device_view::create(single_streams_table, stream);
  rmm::device_uvector<column_device_view> leaf_column_views(0, stream);

  if (num_fragments != 0) {
    // Move column info to device
    col_desc.host_to_device_async(stream);
    leaf_column_views = create_leaf_column_device_views<parquet_column_device_view>(
      col_desc, *parent_column_table_device_view, stream);

    init_row_group_fragments(row_group_fragments,
                             col_desc,
                             partitions,
                             d_part_frag_offset,
                             max_page_fragment_size,
                             stream);
  }

  std::unique_ptr<aggregate_writer_metadata> agg_meta;
  if (!curr_agg_meta) {
    agg_meta = std::make_unique<aggregate_writer_metadata>(
      partitions, kv_meta, this_table_schema, num_columns, stats_granularity);
  } else {
    agg_meta = std::make_unique<aggregate_writer_metadata>(*curr_agg_meta);

    // verify the user isn't passing mismatched tables
    CUDF_EXPECTS(agg_meta->schema_matches(this_table_schema),
                 "Mismatch in schema between multiple calls to write_chunk");

    agg_meta->update_files(partitions);
  }

  auto global_rowgroup_base = agg_meta->num_row_groups_per_file();

  // Decide row group boundaries based on uncompressed data size
  size_type num_rowgroups = 0;

  std::vector<int> num_rg_in_part(partitions.size());
  for (size_t p = 0; p < partitions.size(); ++p) {
    size_type curr_rg_num_rows = 0;
    size_t curr_rg_data_size   = 0;
    int first_frag_in_rg       = part_frag_offset[p];
    int last_frag_in_part      = part_frag_offset[p + 1] - 1;
    for (auto f = first_frag_in_rg; f <= last_frag_in_part; ++f) {
      size_t fragment_data_size = 0;
      for (auto c = 0; c < num_columns; c++) {
        fragment_data_size += row_group_fragments[c][f].fragment_data_size;
      }
      size_type fragment_num_rows = row_group_fragments[0][f].num_rows;

      // If the fragment size gets larger than rg limit then break off a rg
      if (f > first_frag_in_rg &&  // There has to be at least one fragment in row group
          (curr_rg_data_size + fragment_data_size > max_row_group_size ||
           curr_rg_num_rows + fragment_num_rows > max_row_group_rows)) {
        auto& rg    = agg_meta->file(p).row_groups.emplace_back();
        rg.num_rows = curr_rg_num_rows;
        num_rowgroups++;
        num_rg_in_part[p]++;
        curr_rg_num_rows  = 0;
        curr_rg_data_size = 0;
        first_frag_in_rg  = f;
      }
      curr_rg_num_rows += fragment_num_rows;
      curr_rg_data_size += fragment_data_size;

      // TODO: (wishful) refactor to consolidate with above if block
      if (f == last_frag_in_part) {
        auto& rg    = agg_meta->file(p).row_groups.emplace_back();
        rg.num_rows = curr_rg_num_rows;
        num_rowgroups++;
        num_rg_in_part[p]++;
      }
    }
  }

  std::vector<int> first_rg_in_part;
  std::exclusive_scan(
    num_rg_in_part.begin(), num_rg_in_part.end(), std::back_inserter(first_rg_in_part), 0);

  // Initialize row groups and column chunks
  auto const num_chunks = num_rowgroups * num_columns;
  hostdevice_2dvector<EncColumnChunk> chunks(num_rowgroups, num_columns, stream);

  // total fragments per column (in case they are non-uniform)
  std::vector<size_type> frags_per_column(num_columns, 0);

  for (size_t p = 0; p < partitions.size(); ++p) {
    int f               = part_frag_offset[p];
    size_type start_row = partitions[p].start_row;
    for (int r = 0; r < num_rg_in_part[p]; r++) {
      size_t global_r = global_rowgroup_base[p] + r;  // Number of rowgroups already in file/part
      auto& row_group = agg_meta->file(p).row_groups[global_r];
      uint32_t fragments_in_chunk =
        util::div_rounding_up_unsafe(row_group.num_rows, max_page_fragment_size);
      row_group.total_byte_size = 0;
      row_group.columns.resize(num_columns);
      for (int c = 0; c < num_columns; c++) {
        EncColumnChunk& ck = chunks[r + first_rg_in_part[p]][c];

        ck                   = {};
        ck.col_desc          = col_desc.device_ptr() + c;
        ck.col_desc_id       = c;
        ck.fragments         = &row_group_fragments.device_view()[c][f];
        ck.stats             = nullptr;
        ck.start_row         = start_row;
        ck.num_rows          = (uint32_t)row_group.num_rows;
        ck.first_fragment    = c * num_fragments + f;
        ck.encodings         = 0;
        auto chunk_fragments = row_group_fragments[c].subspan(f, fragments_in_chunk);
        // In fragment struct, add a pointer to the chunk it belongs to
        // In each fragment in chunk_fragments, update the chunk pointer here.
        for (auto& frag : chunk_fragments) {
          frag.chunk = &chunks.device_view()[r + first_rg_in_part[p]][c];
        }
        ck.num_values = std::accumulate(
          chunk_fragments.begin(), chunk_fragments.end(), 0, [](uint32_t l, auto r) {
            return l + r.num_values;
          });
        ck.plain_data_size = std::accumulate(
          chunk_fragments.begin(), chunk_fragments.end(), 0, [](int sum, PageFragment frag) {
            return sum + frag.fragment_data_size;
          });
        auto& column_chunk_meta          = row_group.columns[c].meta_data;
        column_chunk_meta.type           = parquet_columns[c].physical_type();
        column_chunk_meta.path_in_schema = parquet_columns[c].get_path_in_schema();
        column_chunk_meta.codec          = UNCOMPRESSED;
        column_chunk_meta.num_values     = ck.num_values;

        frags_per_column[c] += util::div_rounding_up_unsafe(
          row_group.num_rows, std::min(column_frag_size[c], max_page_fragment_size));
      }
      f += fragments_in_chunk;
      start_row += (uint32_t)row_group.num_rows;
    }
  }

  row_group_fragments.host_to_device_async(stream);
  [[maybe_unused]] auto dict_info_owner = build_chunk_dictionaries(
    chunks, col_desc, row_group_fragments, compression, dict_policy, max_dictionary_size, stream);

  // The code preceding this used a uniform fragment size for all columns. Now recompute
  // fragments with a (potentially) varying number of fragments per column.

  // first figure out the total number of fragments and calculate the start offset for each column
  std::vector<size_type> frag_offsets(num_columns, 0);
  std::exclusive_scan(frags_per_column.begin(), frags_per_column.end(), frag_offsets.begin(), 0);
  size_type const total_frags =
    frags_per_column.empty() ? 0 : frag_offsets.back() + frags_per_column.back();

  rmm::device_uvector<statistics_chunk> frag_stats(0, stream);
  cudf::detail::hostdevice_vector<PageFragment> page_fragments(total_frags, stream);

  // update fragments and/or prepare for fragment statistics calculation if necessary
  if (total_frags != 0) {
    if (stats_granularity != statistics_freq::STATISTICS_NONE) {
      frag_stats.resize(total_frags, stream);
    }

    for (int c = 0; c < num_columns; c++) {
      auto frag_offset     = frag_offsets[c];
      auto const frag_size = column_frag_size[c];

      for (size_t p = 0; p < partitions.size(); ++p) {
        for (int r = 0; r < num_rg_in_part[p]; r++) {
          auto const global_r   = global_rowgroup_base[p] + r;
          auto const& row_group = agg_meta->file(p).row_groups[global_r];
          uint32_t const fragments_in_chunk =
            util::div_rounding_up_unsafe(row_group.num_rows, frag_size);
          EncColumnChunk& ck = chunks[r + first_rg_in_part[p]][c];
          ck.fragments       = page_fragments.device_ptr(frag_offset);
          ck.first_fragment  = frag_offset;

          // update the chunk pointer here for each fragment in chunk.fragments
          for (uint32_t i = 0; i < fragments_in_chunk; i++) {
            page_fragments[frag_offset + i].chunk =
              &chunks.device_view()[r + first_rg_in_part[p]][c];
          }

          if (not frag_stats.is_empty()) { ck.stats = frag_stats.data() + frag_offset; }
          frag_offset += fragments_in_chunk;
        }
      }
    }

    chunks.host_to_device_async(stream);

    // re-initialize page fragments
    page_fragments.host_to_device_async(stream);
    calculate_page_fragments(page_fragments, column_frag_size, stream);

    // and gather fragment statistics
    if (not frag_stats.is_empty()) {
      gather_fragment_statistics(frag_stats,
                                 {page_fragments.device_ptr(), static_cast<size_t>(total_frags)},
                                 int96_timestamps,
                                 stream);
    }
  }

  // Build chunk dictionaries and count pages. Sends chunks to device.
  cudf::detail::hostdevice_vector<size_type> comp_page_sizes = init_page_sizes(chunks,
                                                                               col_desc,
                                                                               num_columns,
                                                                               max_page_size_bytes,
                                                                               max_page_size_rows,
                                                                               write_v2_headers,
                                                                               compression,
                                                                               stream);

  // Find which partition a rg belongs to
  std::vector<int> rg_to_part;
  for (size_t p = 0; p < num_rg_in_part.size(); ++p) {
    std::fill_n(std::back_inserter(rg_to_part), num_rg_in_part[p], p);
  }

  // Initialize rowgroups to encode
  size_type num_pages        = 0;
  size_t max_uncomp_bfr_size = 0;
  size_t max_comp_bfr_size   = 0;
  size_t max_chunk_bfr_size  = 0;

  size_t column_index_bfr_size  = 0;
  size_t def_histogram_bfr_size = 0;
  size_t rep_histogram_bfr_size = 0;
  size_t rowgroup_size          = 0;
  size_t comp_rowgroup_size     = 0;
  for (size_type r = 0; r <= num_rowgroups; r++) {
    if (r < num_rowgroups) {
      for (int i = 0; i < num_columns; i++) {
        EncColumnChunk* ck = &chunks[r][i];
        ck->first_page     = num_pages;
        num_pages += ck->num_pages;
        rowgroup_size += ck->bfr_size;
        comp_rowgroup_size += ck->compressed_size;
        max_chunk_bfr_size =
          std::max(max_chunk_bfr_size, (size_t)std::max(ck->bfr_size, ck->compressed_size));
        if (stats_granularity == statistics_freq::STATISTICS_COLUMN) {
          auto const& col = col_desc[ck->col_desc_id];
          column_index_bfr_size += column_index_buffer_size(ck, col, column_index_truncate_length);

          // SizeStatistics are on the ColumnIndex, so only need to allocate the histograms data
          // if we're doing page-level indexes. add 1 to num_pages for per-chunk histograms.
          auto const num_histograms = ck->num_data_pages() + 1;

          if (col.max_def_level > DEF_LVL_HIST_CUTOFF) {
            def_histogram_bfr_size += (col.max_def_level + 1) * num_histograms;
          }
          if (col.max_rep_level > REP_LVL_HIST_CUTOFF) {
            rep_histogram_bfr_size += (col.max_rep_level + 1) * num_histograms;
          }
        }
      }
    }
    // write bfr sizes if this is the last rowgroup
    if (r == num_rowgroups) {
      max_uncomp_bfr_size = rowgroup_size;
      max_comp_bfr_size   = comp_rowgroup_size;
    }
  }

  // Clear compressed buffer size if compression has been turned off
  if (compression == Compression::UNCOMPRESSED) { max_comp_bfr_size = 0; }

  // Initialize data pointers
  uint32_t const num_stats_bfr =
    (stats_granularity != statistics_freq::STATISTICS_NONE) ? num_pages + num_chunks : 0;

  // Buffers need to be padded.
  // Required by `gpuGatherPages`.
  rmm::device_buffer uncomp_bfr(
    cudf::util::round_up_safe(max_uncomp_bfr_size, BUFFER_PADDING_MULTIPLE), stream);
  rmm::device_buffer comp_bfr(cudf::util::round_up_safe(max_comp_bfr_size, BUFFER_PADDING_MULTIPLE),
                              stream);

  rmm::device_buffer col_idx_bfr(column_index_bfr_size, stream);
  rmm::device_uvector<EncPage> pages(num_pages, stream);
  rmm::device_uvector<uint32_t> def_level_histogram(def_histogram_bfr_size, stream);
  rmm::device_uvector<uint32_t> rep_level_histogram(rep_histogram_bfr_size, stream);

  thrust::uninitialized_fill(
    rmm::exec_policy_nosync(stream), def_level_histogram.begin(), def_level_histogram.end(), 0);
  thrust::uninitialized_fill(
    rmm::exec_policy_nosync(stream), rep_level_histogram.begin(), rep_level_histogram.end(), 0);

  // This contains stats for both the pages and the rowgroups. TODO: make them separate.
  rmm::device_uvector<statistics_chunk> page_stats(num_stats_bfr, stream);
  auto bfr_i = static_cast<uint8_t*>(col_idx_bfr.data());
  auto bfr_r = rep_level_histogram.data();
  auto bfr_d = def_level_histogram.data();
  if (num_rowgroups != 0) {
    auto bfr   = static_cast<uint8_t*>(uncomp_bfr.data());
    auto bfr_c = static_cast<uint8_t*>(comp_bfr.data());
    for (auto r = 0; r < num_rowgroups; r++) {
      for (auto i = 0; i < num_columns; i++) {
        EncColumnChunk& ck   = chunks[r][i];
        ck.uncompressed_bfr  = bfr;
        ck.compressed_bfr    = bfr_c;
        ck.column_index_blob = bfr_i;
        bfr += ck.bfr_size;
        bfr_c += ck.compressed_size;
        if (stats_granularity == statistics_freq::STATISTICS_COLUMN) {
          auto const& col      = col_desc[ck.col_desc_id];
          ck.column_index_size = column_index_buffer_size(&ck, col, column_index_truncate_length);
          bfr_i += ck.column_index_size;

          auto const num_histograms = ck.num_data_pages() + 1;
          if (col.max_def_level > DEF_LVL_HIST_CUTOFF) {
            ck.def_histogram_data = bfr_d;
            bfr_d += num_histograms * (col.max_def_level + 1);
          }
          if (col.max_rep_level > REP_LVL_HIST_CUTOFF) {
            ck.rep_histogram_data = bfr_r;
            bfr_r += num_histograms * (col.max_rep_level + 1);
          }
        }
      }
    }
  }

  if (num_pages != 0) {
    init_encoder_pages(chunks,
                       col_desc,
                       {pages.data(), pages.size()},
                       comp_page_sizes,
                       (num_stats_bfr) ? page_stats.data() : nullptr,
                       (num_stats_bfr) ? frag_stats.data() : nullptr,
                       num_columns,
                       num_pages,
                       num_stats_bfr,
                       compression,
                       max_page_size_bytes,
                       max_page_size_rows,
                       write_v2_headers,
                       stream);
  }

  // Check device write support for all chunks and initialize bounce_buffer.
  bool all_device_write   = true;
  uint32_t max_write_size = 0;
  std::optional<writer_compression_statistics> comp_stats;
  if (collect_compression_statistics) { comp_stats = writer_compression_statistics{}; }

  // Encode row groups
  if (num_rowgroups != 0) {
    encode_pages(
      chunks,
      {pages.data(), pages.size()},
      (stats_granularity == statistics_freq::STATISTICS_PAGE) ? page_stats.data() : nullptr,
      (stats_granularity != statistics_freq::STATISTICS_NONE) ? page_stats.data() + num_pages
                                                              : nullptr,
      (stats_granularity == statistics_freq::STATISTICS_COLUMN) ? page_stats.data() : nullptr,
      comp_stats,
      compression,
      column_index_truncate_length,
      write_v2_headers,
      stream);

    bool need_sync{false};

    // need to fetch the histogram data from the device
    std::vector<uint32_t> h_def_histogram;
    std::vector<uint32_t> h_rep_histogram;
    if (stats_granularity == statistics_freq::STATISTICS_COLUMN) {
      if (def_histogram_bfr_size > 0) {
        h_def_histogram =
          std::move(cudf::detail::make_std_vector_async(def_level_histogram, stream));
        need_sync = true;
      }
      if (rep_histogram_bfr_size > 0) {
        h_rep_histogram =
          std::move(cudf::detail::make_std_vector_async(rep_level_histogram, stream));
        need_sync = true;
      }
    }

    for (int r = 0; r < num_rowgroups; r++) {
      int p           = rg_to_part[r];
      int global_r    = global_rowgroup_base[p] + r - first_rg_in_part[p];
      auto& row_group = agg_meta->file(p).row_groups[global_r];

      for (auto i = 0; i < num_columns; i++) {
        auto const& ck          = chunks[r][i];
        auto const dev_bfr      = ck.is_compressed ? ck.compressed_bfr : ck.uncompressed_bfr;
        auto& column_chunk_meta = row_group.columns[i].meta_data;

        if (ck.is_compressed) { column_chunk_meta.codec = compression; }
        if (!out_sink[p]->is_device_write_preferred(ck.compressed_size)) {
          all_device_write = false;
        }
        max_write_size = std::max(max_write_size, ck.compressed_size);

        update_chunk_encodings(column_chunk_meta.encodings, ck.encodings);
        update_chunk_encoding_stats(column_chunk_meta, ck, write_v2_headers);

        if (ck.ck_stat_size != 0) {
          std::vector<uint8_t> const stats_blob = cudf::detail::make_std_vector_sync(
            device_span<uint8_t const>(dev_bfr, ck.ck_stat_size), stream);
          CompactProtocolReader cp(stats_blob.data(), stats_blob.size());
          cp.read(&column_chunk_meta.statistics);
          need_sync = true;
        }

        row_group.total_byte_size += ck.bfr_size;
        row_group.total_compressed_size =
          row_group.total_compressed_size.value_or(0) + ck.compressed_size;
        column_chunk_meta.total_uncompressed_size = ck.bfr_size;
        column_chunk_meta.total_compressed_size   = ck.compressed_size;
      }
    }

    // Sync before calling the next `encode_pages` which may alter the stats data.
    if (need_sync) { stream.synchronize(); }

    // now add to the column chunk SizeStatistics if necessary
    if (stats_granularity == statistics_freq::STATISTICS_COLUMN) {
      auto h_def_ptr = h_def_histogram.data();
      auto h_rep_ptr = h_rep_histogram.data();

      for (int r = 0; r < num_rowgroups; r++) {
        int const p        = rg_to_part[r];
        int const global_r = global_rowgroup_base[p] + r - first_rg_in_part[p];
        auto& row_group    = agg_meta->file(p).row_groups[global_r];

        for (auto i = 0; i < num_columns; i++) {
          auto const& ck          = chunks[r][i];
          auto const& col         = col_desc[ck.col_desc_id];
          auto& column_chunk_meta = row_group.columns[i].meta_data;

          // Add SizeStatistics for the chunk. For now we're only going to do the column chunk
          // stats if we're also doing them at the page level. There really isn't much value for
          // us in per-chunk stats since everything we do processing wise is at the page level.
          SizeStatistics chunk_stats;

          // var_byte_size will only be non-zero for byte array columns.
          if (ck.var_bytes_size > 0) {
            chunk_stats.unencoded_byte_array_data_bytes = ck.var_bytes_size;
          }

          auto const num_data_pages = ck.num_data_pages();
          if (col.max_def_level > DEF_LVL_HIST_CUTOFF) {
            size_t const hist_size        = col.max_def_level + 1;
            uint32_t const* const ck_hist = h_def_ptr + hist_size * num_data_pages;
            host_span<uint32_t const> ck_def_hist{ck_hist, hist_size};

            chunk_stats.definition_level_histogram = {ck_def_hist.begin(), ck_def_hist.end()};
            h_def_ptr += hist_size * (num_data_pages + 1);
          }

          if (col.max_rep_level > REP_LVL_HIST_CUTOFF) {
            size_t const hist_size        = col.max_rep_level + 1;
            uint32_t const* const ck_hist = h_rep_ptr + hist_size * num_data_pages;
            host_span<uint32_t const> ck_rep_hist{ck_hist, hist_size};

            chunk_stats.repetition_level_histogram = {ck_rep_hist.begin(), ck_rep_hist.end()};
            h_rep_ptr += hist_size * (num_data_pages + 1);
          }

          if (chunk_stats.unencoded_byte_array_data_bytes.has_value() ||
              chunk_stats.definition_level_histogram.has_value() ||
              chunk_stats.repetition_level_histogram.has_value()) {
            column_chunk_meta.size_statistics = std::move(chunk_stats);
          }
        }
      }
    }
  }

  auto bounce_buffer =
    cudf::detail::pinned_host_vector<uint8_t>(all_device_write ? 0 : max_write_size);

  return std::tuple{std::move(agg_meta),
                    std::move(pages),
                    std::move(chunks),
                    std::move(global_rowgroup_base),
                    std::move(first_rg_in_part),
                    std::move(rg_to_part),
                    std::move(comp_stats),
                    std::move(uncomp_bfr),
                    std::move(comp_bfr),
                    std::move(col_idx_bfr),
                    std::move(bounce_buffer)};
}

}  // namespace

writer::impl::impl(std::vector<std::unique_ptr<data_sink>> sinks,
                   parquet_writer_options const& options,
                   single_write_mode mode,
                   rmm::cuda_stream_view stream)
  : _stream(stream),
    _compression(to_parquet_compression(options.get_compression())),
    _max_row_group_size{options.get_row_group_size_bytes()},
    _max_row_group_rows{options.get_row_group_size_rows()},
    _max_page_size_bytes(max_page_bytes(_compression, options.get_max_page_size_bytes())),
    _max_page_size_rows(options.get_max_page_size_rows()),
    _stats_granularity(options.get_stats_level()),
    _dict_policy(options.get_dictionary_policy()),
    _max_dictionary_size(options.get_max_dictionary_size()),
    _max_page_fragment_size(options.get_max_page_fragment_size()),
    _int96_timestamps(options.is_enabled_int96_timestamps()),
    _utc_timestamps(options.is_enabled_utc_timestamps()),
    _write_v2_headers(options.is_enabled_write_v2_headers()),
    _sorting_columns(options.get_sorting_columns()),
    _column_index_truncate_length(options.get_column_index_truncate_length()),
    _kv_meta(options.get_key_value_metadata()),
    _single_write_mode(mode),
    _out_sink(std::move(sinks)),
    _compression_statistics{options.get_compression_statistics()}
{
  if (options.get_metadata()) {
    _table_meta = std::make_unique<table_input_metadata>(*options.get_metadata());
  }
  init_state();
}

writer::impl::impl(std::vector<std::unique_ptr<data_sink>> sinks,
                   chunked_parquet_writer_options const& options,
                   single_write_mode mode,
                   rmm::cuda_stream_view stream)
  : _stream(stream),
    _compression(to_parquet_compression(options.get_compression())),
    _max_row_group_size{options.get_row_group_size_bytes()},
    _max_row_group_rows{options.get_row_group_size_rows()},
    _max_page_size_bytes(max_page_bytes(_compression, options.get_max_page_size_bytes())),
    _max_page_size_rows(options.get_max_page_size_rows()),
    _stats_granularity(options.get_stats_level()),
    _dict_policy(options.get_dictionary_policy()),
    _max_dictionary_size(options.get_max_dictionary_size()),
    _max_page_fragment_size(options.get_max_page_fragment_size()),
    _int96_timestamps(options.is_enabled_int96_timestamps()),
    _utc_timestamps(options.is_enabled_utc_timestamps()),
    _write_v2_headers(options.is_enabled_write_v2_headers()),
    _sorting_columns(options.get_sorting_columns()),
    _column_index_truncate_length(options.get_column_index_truncate_length()),
    _kv_meta(options.get_key_value_metadata()),
    _single_write_mode(mode),
    _out_sink(std::move(sinks)),
    _compression_statistics{options.get_compression_statistics()}
{
  if (options.get_metadata()) {
    _table_meta = std::make_unique<table_input_metadata>(*options.get_metadata());
  }
  init_state();
}

writer::impl::~impl() { close(); }

void writer::impl::init_state()
{
  _current_chunk_offset.resize(_out_sink.size());
  // Write file header
  file_header_s fhdr;
  fhdr.magic = parquet_magic;
  for (auto& sink : _out_sink) {
    sink->host_write(&fhdr, sizeof(fhdr));
  }
  std::fill_n(_current_chunk_offset.begin(), _current_chunk_offset.size(), sizeof(file_header_s));
}

void writer::impl::update_compression_statistics(
  std::optional<writer_compression_statistics> const& compression_stats)
{
  if (compression_stats.has_value() and _compression_statistics != nullptr) {
    *_compression_statistics += compression_stats.value();
  }
}

void writer::impl::write(table_view const& input, std::vector<partition_info> const& partitions)
{
  _last_write_successful = false;
  CUDF_EXPECTS(not _closed, "Data has already been flushed to out and closed");

  if (not _table_meta) { _table_meta = std::make_unique<table_input_metadata>(input); }
  fill_table_meta(_table_meta);

  // All kinds of memory allocation and data compressions/encoding are performed here.
  // If any error occurs, such as out-of-memory exception, the internal state of the current
  // writer is still intact.
  [[maybe_unused]] auto [updated_agg_meta,
                         pages,
                         chunks,
                         global_rowgroup_base,
                         first_rg_in_part,
                         rg_to_part,
                         comp_stats,
                         uncomp_bfr,   // unused, but contains data for later write to sink
                         comp_bfr,     // unused, but contains data for later write to sink
                         col_idx_bfr,  // unused, but contains data for later write to sink
                         bounce_buffer] = [&] {
    try {
      return convert_table_to_parquet_data(*_table_meta,
                                           input,
                                           partitions,
                                           _kv_meta,
                                           _agg_meta,
                                           _max_page_fragment_size,
                                           _max_row_group_size,
                                           _max_page_size_bytes,
                                           _max_row_group_rows,
                                           _max_page_size_rows,
                                           _column_index_truncate_length,
                                           _stats_granularity,
                                           _compression,
                                           _compression_statistics != nullptr,
                                           _dict_policy,
                                           _max_dictionary_size,
                                           _single_write_mode,
                                           _int96_timestamps,
                                           _utc_timestamps,
                                           _write_v2_headers,
                                           _out_sink,
                                           _stream);
    } catch (...) {  // catch any exception type
      CUDF_LOG_ERROR(
        "Parquet writer encountered exception during processing. "
        "No data has been written to the sink.");
      throw;  // this throws the same exception
    }
  }();

  // Compression/encoding were all successful. Now write the intermediate results.
  write_parquet_data_to_sink(updated_agg_meta,
                             pages,
                             chunks,
                             global_rowgroup_base,
                             first_rg_in_part,
                             rg_to_part,
                             bounce_buffer);

  update_compression_statistics(comp_stats);

  _last_write_successful = true;
}

void writer::impl::write_parquet_data_to_sink(
  std::unique_ptr<aggregate_writer_metadata>& updated_agg_meta,
  device_span<EncPage const> pages,
  host_2dspan<EncColumnChunk const> chunks,
  host_span<size_t const> global_rowgroup_base,
  host_span<int const> first_rg_in_part,
  host_span<int const> rg_to_part,
  host_span<uint8_t> bounce_buffer)
{
  _agg_meta                = std::move(updated_agg_meta);
  auto const num_rowgroups = chunks.size().first;
  auto const num_columns   = chunks.size().second;

  if (num_rowgroups != 0) {
    std::vector<std::future<void>> write_tasks;

    for (auto r = 0; r < static_cast<int>(num_rowgroups); r++) {
      int const p        = rg_to_part[r];
      int const global_r = global_rowgroup_base[p] + r - first_rg_in_part[p];
      auto& row_group    = _agg_meta->file(p).row_groups[global_r];

      for (std::size_t i = 0; i < num_columns; i++) {
        auto const& ck     = chunks[r][i];
        auto const dev_bfr = ck.is_compressed ? ck.compressed_bfr : ck.uncompressed_bfr;

        // Skip the range [0, ck.ck_stat_size) since it has already been copied to host
        // and stored in _agg_meta before.
        if (_out_sink[p]->is_device_write_preferred(ck.compressed_size)) {
          write_tasks.push_back(_out_sink[p]->device_write_async(
            dev_bfr + ck.ck_stat_size, ck.compressed_size, _stream));
        } else {
          CUDF_EXPECTS(bounce_buffer.size() >= ck.compressed_size,
                       "Bounce buffer was not properly initialized.");
          CUDF_CUDA_TRY(hipMemcpyAsync(bounce_buffer.data(),
                                        dev_bfr + ck.ck_stat_size,
                                        ck.compressed_size,
                                        hipMemcpyDefault,
                                        _stream.value()));
          _stream.synchronize();
          _out_sink[p]->host_write(bounce_buffer.data(), ck.compressed_size);
        }

        auto const chunk_offset = _current_chunk_offset[p];
        auto& column_chunk_meta = row_group.columns[i].meta_data;
        column_chunk_meta.data_page_offset =
          chunk_offset + ((ck.use_dictionary) ? ck.dictionary_size : 0);
        column_chunk_meta.dictionary_page_offset = (ck.use_dictionary) ? chunk_offset : 0;
        _current_chunk_offset[p] += ck.compressed_size;

        // save location of first page in row group
        if (i == 0) { row_group.file_offset = chunk_offset; }
      }
    }
    for (auto const& task : write_tasks) {
      task.wait();
    }
  }

  if (_stats_granularity == statistics_freq::STATISTICS_COLUMN) {
    // need pages on host to create offset_indexes
    auto const h_pages = cudf::detail::make_host_vector_sync(pages, _stream);

    // add column and offset indexes to metadata
    if (num_rowgroups != 0) {
      auto curr_page_idx = chunks[0][0].first_page;
      for (auto r = 0; r < static_cast<int>(num_rowgroups); r++) {
        int const p           = rg_to_part[r];
        int const global_r    = global_rowgroup_base[p] + r - first_rg_in_part[p];
        auto const& row_group = _agg_meta->file(p).row_groups[global_r];
        for (std::size_t i = 0; i < num_columns; i++) {
          EncColumnChunk const& ck      = chunks[r][i];
          auto const& column_chunk_meta = row_group.columns[i].meta_data;

          // start transfer of the column index
          std::vector<uint8_t> column_idx;
          column_idx.resize(ck.column_index_size);
          CUDF_CUDA_TRY(hipMemcpyAsync(column_idx.data(),
                                        ck.column_index_blob,
                                        ck.column_index_size,
                                        hipMemcpyDefault,
                                        _stream.value()));

          // calculate offsets while the column index is transferring
          int64_t curr_pg_offset = column_chunk_meta.data_page_offset;

          OffsetIndex offset_idx;
          std::vector<int64_t> var_bytes;
          auto const is_byte_arr = column_chunk_meta.type == BYTE_ARRAY;

          for (uint32_t pg = 0; pg < ck.num_pages; pg++) {
            auto const& enc_page = h_pages[curr_page_idx++];

            // skip dict pages
            if (enc_page.page_type == PageType::DICTIONARY_PAGE) { continue; }

            int32_t const this_page_size =
              enc_page.hdr_size + (ck.is_compressed ? enc_page.comp_data_size : enc_page.data_size);
            // first_row_idx is relative to start of row group
            PageLocation loc{curr_pg_offset, this_page_size, enc_page.start_row - ck.start_row};
            if (is_byte_arr) { var_bytes.push_back(enc_page.var_bytes_size); }
            offset_idx.page_locations.push_back(loc);
            curr_pg_offset += this_page_size;
          }

          if (is_byte_arr) { offset_idx.unencoded_byte_array_data_bytes = std::move(var_bytes); }

          _stream.synchronize();
          _agg_meta->file(p).offset_indexes.emplace_back(std::move(offset_idx));
          _agg_meta->file(p).column_indexes.emplace_back(std::move(column_idx));
        }
      }
    }
  }
}

std::unique_ptr<std::vector<uint8_t>> writer::impl::close(
  std::vector<std::string> const& column_chunks_file_path)
{
  if (_closed) { return nullptr; }
  _closed = true;
  if (not _last_write_successful) { return nullptr; }
  for (size_t p = 0; p < _out_sink.size(); p++) {
    std::vector<uint8_t> buffer;
    CompactProtocolWriter cpw(&buffer);
    file_ender_s fendr;
    auto& fmd = _agg_meta->file(p);

    if (_stats_granularity == statistics_freq::STATISTICS_COLUMN) {
      // write column indices, updating column metadata along the way
      int chunkidx = 0;
      for (auto& r : fmd.row_groups) {
        for (auto& c : r.columns) {
          auto const& index     = fmd.column_indexes[chunkidx++];
          c.column_index_offset = _out_sink[p]->bytes_written();
          c.column_index_length = index.size();
          _out_sink[p]->host_write(index.data(), index.size());
        }
      }

      // write offset indices, updating column metadata along the way
      chunkidx = 0;
      for (auto& r : fmd.row_groups) {
        for (auto& c : r.columns) {
          auto const& offsets = fmd.offset_indexes[chunkidx++];
          buffer.resize(0);
          int32_t len           = cpw.write(offsets);
          c.offset_index_offset = _out_sink[p]->bytes_written();
          c.offset_index_length = len;
          _out_sink[p]->host_write(buffer.data(), buffer.size());
        }
      }
    }

    // set row group ordinals
    auto iter        = thrust::make_counting_iterator(0);
    auto& row_groups = fmd.row_groups;
    std::for_each(
      iter, iter + row_groups.size(), [&row_groups](auto idx) { row_groups[idx].ordinal = idx; });

    // set sorting_columns on row groups
    if (_sorting_columns.has_value()) {
      // convert `sorting_column` to `SortingColumn`
      auto const& sorting_cols = _sorting_columns.value();
      std::vector<SortingColumn> scols;
      std::transform(
        sorting_cols.begin(), sorting_cols.end(), std::back_inserter(scols), [](auto const& sc) {
          return SortingColumn{sc.column_idx, sc.is_descending, sc.is_nulls_first};
        });
      // and copy to each row group
      std::for_each(iter, iter + row_groups.size(), [&row_groups, &scols](auto idx) {
        row_groups[idx].sorting_columns = scols;
      });
    }
    buffer.resize(0);
    fendr.footer_len = static_cast<uint32_t>(cpw.write(_agg_meta->get_metadata(p)));
    fendr.magic      = parquet_magic;
    _out_sink[p]->host_write(buffer.data(), buffer.size());
    _out_sink[p]->host_write(&fendr, sizeof(fendr));
    _out_sink[p]->flush();
  }

  // Optionally output raw file metadata with the specified column chunk file path
  if (column_chunks_file_path.size() > 0) {
    CUDF_EXPECTS(column_chunks_file_path.size() == _agg_meta->num_files(),
                 "Expected one column chunk path per output file");
    _agg_meta->set_file_paths(column_chunks_file_path);
    file_header_s fhdr = {parquet_magic};
    std::vector<uint8_t> buffer;
    CompactProtocolWriter cpw(&buffer);
    buffer.insert(buffer.end(),
                  reinterpret_cast<uint8_t const*>(&fhdr),
                  reinterpret_cast<uint8_t const*>(&fhdr) + sizeof(fhdr));
    file_ender_s fendr;
    fendr.magic      = parquet_magic;
    fendr.footer_len = static_cast<uint32_t>(cpw.write(_agg_meta->get_merged_metadata()));
    buffer.insert(buffer.end(),
                  reinterpret_cast<uint8_t const*>(&fendr),
                  reinterpret_cast<uint8_t const*>(&fendr) + sizeof(fendr));
    return std::make_unique<std::vector<uint8_t>>(std::move(buffer));
  } else {
    return {nullptr};
  }
  return nullptr;
}

// Forward to implementation
writer::writer(std::vector<std::unique_ptr<data_sink>> sinks,
               parquet_writer_options const& options,
               single_write_mode mode,
               rmm::cuda_stream_view stream)
  : _impl(std::make_unique<impl>(std::move(sinks), options, mode, stream))
{
}

writer::writer(std::vector<std::unique_ptr<data_sink>> sinks,
               chunked_parquet_writer_options const& options,
               single_write_mode mode,
               rmm::cuda_stream_view stream)
  : _impl(std::make_unique<impl>(std::move(sinks), options, mode, stream))
{
}

// Destructor within this translation unit
writer::~writer() = default;

// Forward to implementation
void writer::write(table_view const& table, std::vector<partition_info> const& partitions)
{
  _impl->write(
    table, partitions.empty() ? std::vector<partition_info>{{0, table.num_rows()}} : partitions);
}

// Forward to implementation
std::unique_ptr<std::vector<uint8_t>> writer::close(
  std::vector<std::string> const& column_chunks_file_path)
{
  return _impl->close(column_chunks_file_path);
}

std::unique_ptr<std::vector<uint8_t>> writer::merge_row_group_metadata(
  std::vector<std::unique_ptr<std::vector<uint8_t>>> const& metadata_list)
{
  std::vector<uint8_t> output;
  CompactProtocolWriter cpw(&output);
  FileMetaData md;

  md.row_groups.reserve(metadata_list.size());
  for (auto const& blob : metadata_list) {
    CompactProtocolReader cpreader(
      blob.get()->data(),
      std::max<size_t>(blob.get()->size(), sizeof(file_ender_s)) - sizeof(file_ender_s));
    cpreader.skip_bytes(sizeof(file_header_s));  // Skip over file header
    if (md.num_rows == 0) {
      cpreader.read(&md);
    } else {
      FileMetaData tmp;
      cpreader.read(&tmp);
      md.row_groups.insert(md.row_groups.end(),
                           std::make_move_iterator(tmp.row_groups.begin()),
                           std::make_move_iterator(tmp.row_groups.end()));
      md.num_rows += tmp.num_rows;
    }
  }

  // Remove any LogicalType::UNKNOWN annotations that were passed in as they can confuse
  // column type inferencing.
  // See https://github.com/rapidsai/cudf/pull/14264#issuecomment-1778311615
  for (auto& se : md.schema) {
    if (se.logical_type.has_value() && se.logical_type.value().type == LogicalType::UNKNOWN) {
      se.logical_type = thrust::nullopt;
    }
  }

  // Thrift-encode the resulting output
  file_header_s fhdr;
  file_ender_s fendr;
  fhdr.magic = parquet_magic;
  output.insert(output.end(),
                reinterpret_cast<uint8_t const*>(&fhdr),
                reinterpret_cast<uint8_t const*>(&fhdr) + sizeof(fhdr));
  fendr.footer_len = static_cast<uint32_t>(cpw.write(md));
  fendr.magic      = parquet_magic;
  output.insert(output.end(),
                reinterpret_cast<uint8_t const*>(&fendr),
                reinterpret_cast<uint8_t const*>(&fendr) + sizeof(fendr));
  return std::make_unique<std::vector<uint8_t>>(std::move(output));
}

}  // namespace cudf::io::parquet::detail
