#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2025, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "hybrid_scan_helpers.hpp"
#include "hybrid_scan_impl.hpp"
#include "io/parquet/reader_impl_preprocess_utils.cuh"
#include "io/utilities/time_utils.cuh"

#include <cudf/detail/iterator.cuh>
#include <cudf/detail/nvtx/ranges.hpp>
#include <cudf/detail/utilities/batched_memset.hpp>
#include <cudf/detail/utilities/vector_factories.hpp>
#include <cudf/io/parquet_schema.hpp>
#include <cudf/types.hpp>
#include <cudf/utilities/memory_resource.hpp>

#include <rmm/cuda_stream_view.hpp>
#include <rmm/exec_policy.hpp>

#include <cuda/functional>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/reduce.h>

#include <numeric>

namespace cudf::io::parquet::experimental::detail {

namespace {

using parquet::detail::chunk_page_info;
using parquet::detail::ColumnChunkDesc;
using parquet::detail::PageInfo;

/**
 * @brief Decode the dictionary page information from each column chunk
 *
 * @param chunks Host device span of column chunk descriptors, one per input column chunk
 * @param pages Host device span of empty page headers to fill in, one per input column chunk
 * @param stream CUDA stream
 */
void decode_dictionary_page_headers(cudf::detail::hostdevice_span<ColumnChunkDesc> chunks,
                                    cudf::detail::hostdevice_span<PageInfo> pages,
                                    rmm::cuda_stream_view stream)
{
  CUDF_FUNC_RANGE();

  std::vector<size_t> host_chunk_page_counts(chunks.size() + 1);
  std::transform(
    chunks.host_begin(), chunks.host_end(), host_chunk_page_counts.begin(), [](auto const& chunk) {
      return chunk.num_dict_pages;
    });
  host_chunk_page_counts[chunks.size()] = 0;

  auto chunk_page_counts = cudf::detail::make_device_uvector_async(
    host_chunk_page_counts, stream, cudf::get_current_device_resource_ref());

  thrust::exclusive_scan(rmm::exec_policy_nosync(stream),
                         chunk_page_counts.begin(),
                         chunk_page_counts.end(),
                         chunk_page_counts.begin(),
                         size_t{0},
                         cuda::std::plus<size_t>{});

  rmm::device_uvector<chunk_page_info> d_chunk_page_info(chunks.size(), stream);

  thrust::for_each(rmm::exec_policy_nosync(stream),
                   thrust::counting_iterator<cuda::std::size_t>(0),
                   thrust::counting_iterator(chunks.size()),
                   [cpi               = d_chunk_page_info.begin(),
                    chunk_page_counts = chunk_page_counts.begin(),
                    pages             = pages.device_begin()] __device__(size_t i) {
                     cpi[i].pages = &pages[chunk_page_counts[i]];
                   });

  parquet::kernel_error error_code(stream);

  parquet::detail::decode_page_headers(
    chunks.device_begin(), d_chunk_page_info.begin(), chunks.size(), error_code.data(), stream);

  if (auto const error = error_code.value_sync(stream); error != 0) {
    CUDF_FAIL("Parquet header parsing failed with code(s) " +
              parquet::kernel_error::to_string(error));
  }

  // Setup dictionary page for each chunk
  thrust::for_each(rmm::exec_policy_nosync(stream),
                   pages.device_begin(),
                   pages.device_end(),
                   [chunks = chunks.device_begin()] __device__(PageInfo const& p) {
                     if (p.flags & parquet::detail::PAGEINFO_FLAGS_DICTIONARY) {
                       chunks[p.chunk_idx].dict_page = &p;
                     }
                   });

  pages.device_to_host_async(stream);
  chunks.device_to_host_async(stream);
  stream.synchronize();
}

}  // namespace

void hybrid_scan_reader_impl::prepare_row_groups(
  read_mode mode, cudf::host_span<std::vector<size_type> const> row_group_indices)
{
  std::tie(_file_itm_data.global_skip_rows,
           _file_itm_data.global_num_rows,
           _file_itm_data.row_groups,
           _file_itm_data.num_rows_per_source,
           _file_itm_data.num_input_row_groups,
           _file_itm_data.surviving_row_groups) =
    _extended_metadata->select_row_groups({}, row_group_indices, {}, {}, {}, {}, {}, _stream);

  CUDF_EXPECTS(
    std::cmp_less_equal(_file_itm_data.global_num_rows, std::numeric_limits<size_type>::max()),
    "READ_ALL mode does not support reading number of rows more than cudf's column size limit. "
    "For reading larger number of rows, please use chunked_parquet_reader.",
    std::overflow_error);

  // check for page indexes
  _has_page_index = std::all_of(_file_itm_data.row_groups.cbegin(),
                                _file_itm_data.row_groups.cend(),
                                [](auto const& row_group) { return row_group.has_page_index(); });

  if (_file_itm_data.global_num_rows > 0 && not _file_itm_data.row_groups.empty() &&
      not _input_columns.empty()) {
    // fills in chunk information without physically loading or decompressing
    // the associated data
    create_global_chunk_info();

    // compute schedule of input reads.
    compute_input_passes(mode);
  }

  _file_preprocessed = true;
}

bool hybrid_scan_reader_impl::setup_column_chunks()
{
  auto const& row_groups_info = _pass_itm_data->row_groups;
  auto& chunks                = _pass_itm_data->chunks;

  // Descriptors for all the chunks that make up the selected columns
  auto const num_input_columns = _input_columns.size();
  auto const num_chunks        = row_groups_info.size() * num_input_columns;

  // Initialize column chunk information
  size_t total_decompressed_size = 0;
  size_type chunk_count          = 0;
  for (auto const& rg : row_groups_info) {
    // generate ColumnChunkDesc objects for everything to be decoded (all input columns)
    for (size_t i = 0; i < num_input_columns; ++i) {
      auto const& col = _input_columns[i];
      // look up metadata
      auto& col_meta =
        _extended_metadata->get_column_metadata(rg.index, rg.source_index, col.schema_idx);

      if (col_meta.codec != Compression::UNCOMPRESSED) {
        total_decompressed_size += col_meta.total_uncompressed_size;
      }

      // Set pointer to compressed data
      chunks[chunk_count].compressed_data =
        static_cast<uint8_t const*>(_pass_itm_data->raw_page_data[chunk_count].data());

      chunk_count++;
    }
  }
  return total_decompressed_size > 0;
}

void hybrid_scan_reader_impl::setup_compressed_data(
  std::vector<rmm::device_buffer> column_chunk_buffers)
{
  auto& pass = *_pass_itm_data;

  // This function should never be called if `num_rows == 0`.
  CUDF_EXPECTS(_pass_itm_data->num_rows > 0, "Number of reading rows must not be zero.");

  auto& chunks = pass.chunks;

  // Move column chunk buffers to raw page data.
  _pass_itm_data->raw_page_data = std::move(column_chunk_buffers);

  pass.has_compressed_data = setup_column_chunks();

  // Process dataset chunk pages into output columns
  auto const total_pages = _has_page_index ? count_page_headers_with_pgidx(chunks, _stream)
                                           : count_page_headers(chunks, _stream);
  if (total_pages <= 0) { return; }
  rmm::device_uvector<PageInfo> unsorted_pages(total_pages, _stream);

  // decoding of column/page information
  parquet::detail::decode_page_headers(pass, unsorted_pages, _has_page_index, _stream);
  CUDF_EXPECTS(pass.page_offsets.size() - 1 == static_cast<size_t>(_input_columns.size()),
               "Encountered page_offsets / num_columns mismatch");
}

std::tuple<bool,
           cudf::detail::hostdevice_vector<ColumnChunkDesc>,
           cudf::detail::hostdevice_vector<PageInfo>>
hybrid_scan_reader_impl::prepare_dictionaries(
  cudf::host_span<std::vector<size_type> const> row_group_indices,
  cudf::host_span<rmm::device_buffer> dictionary_page_data,
  cudf::host_span<int const> dictionary_col_schemas,
  parquet_reader_options const& options,
  rmm::cuda_stream_view stream)
{
  // Create row group information for the input row group indices
  auto const row_groups_info = std::get<2>(
    _extended_metadata->select_row_groups({}, row_group_indices, {}, {}, {}, {}, {}, _stream));

  CUDF_EXPECTS(row_groups_info.size() * _input_columns.size() == dictionary_page_data.size(),
               "Dictionary page data size must match the number of row groups times the number of "
               "input columns");

  // Number of input columns
  auto const num_input_columns = _input_columns.size();
  // Number of column chunks
  auto const total_column_chunks = dictionary_page_data.size();

  // Boolean to check if any of the column chunnks have compressed data
  auto has_compressed_data = false;

  // Initialize column chunk descriptors
  auto chunks = cudf::detail::hostdevice_vector<cudf::io::parquet::detail::ColumnChunkDesc>(
    total_column_chunks, stream);
  auto chunk_idx = 0;

  // For all row groups
  for (auto const& rg : row_groups_info) {
    auto const& row_group = _extended_metadata->get_row_group(rg.index, rg.source_index);

    // For all columns with dictionary page and (in)equality predicate
    for (auto col_schema_idx : dictionary_col_schemas) {
      // look up metadata
      auto& col_meta =
        _extended_metadata->get_column_metadata(rg.index, rg.source_index, col_schema_idx);
      auto& schema = _extended_metadata->get_schema(
        _extended_metadata->map_schema_index(col_schema_idx, rg.source_index), rg.source_index);

      // dictionary data buffer for this column chunk
      auto& dict_page_data = dictionary_page_data[chunk_idx];

      // Check if the column chunk has compressed data
      has_compressed_data |=
        col_meta.codec != Compression::UNCOMPRESSED and col_meta.total_compressed_size > 0;

      // TODO: Use `parquet::detail::conversion_info` instead of directly computing `clock_rate`
      // when AST support for decimals is available
      auto const column_type_id =
        parquet::detail::to_type_id(schema,
                                    options.is_enabled_convert_strings_to_categories(),
                                    options.get_timestamp_type().id());
      auto const clock_rate = is_chrono(data_type{column_type_id})
                                ? to_clockrate(options.get_timestamp_type().id())
                                : int32_t{0};

      // Create a column chunk descriptor - zero/null values for all fields that are not needed
      chunks[chunk_idx] = ColumnChunkDesc(static_cast<int64_t>(dict_page_data.size()),
                                          static_cast<uint8_t*>(dict_page_data.data()),
                                          col_meta.num_values,
                                          schema.type,
                                          schema.type_length,
                                          0,  // start_row
                                          0,  // num_rows
                                          0,  // max_definition_level
                                          0,  // max_repetition_level
                                          0,  // max_nesting_depth
                                          0,  // def_level_bits
                                          0,  // rep_level_bits
                                          col_meta.codec,
                                          schema.logical_type,
                                          clock_rate,
                                          0,  // src_col_index
                                          col_schema_idx,
                                          nullptr,  // chunk_info
                                          0.0f,     // list_bytes_per_row_est
                                          false,    // strings_to_categorical
                                          rg.source_index);
      // Set the number of dictionary and data pages
      chunks[chunk_idx].num_dict_pages = static_cast<int32_t>(dict_page_data.size() > 0);
      chunks[chunk_idx].num_data_pages = 0;  // Always zero at this stage
      chunk_idx++;
    }
  }

  // Copy the column chunk descriptors to the device
  chunks.host_to_device_async(stream);

  // Create page infos for each column chunk's dictionary page
  cudf::detail::hostdevice_vector<PageInfo> pages(total_column_chunks, stream);

  // Decode dictionary page headers
  decode_dictionary_page_headers(chunks, pages, stream);

  return {has_compressed_data, std::move(chunks), std::move(pages)};
}

void hybrid_scan_reader_impl::update_row_mask(cudf::column_view const& in_row_mask,
                                              cudf::mutable_column_view& out_row_mask,
                                              cudf::size_type out_row_mask_offset,
                                              rmm::cuda_stream_view stream)
{
  CUDF_FUNC_RANGE();

  auto const total_rows = static_cast<cudf::size_type>(in_row_mask.size());

  CUDF_EXPECTS(out_row_mask_offset + total_rows <= out_row_mask.size(),
               "Input and output row mask columns must have the same number of rows");
  CUDF_EXPECTS(out_row_mask.type().id() == type_id::BOOL8,
               "Output row mask column must be a boolean column");
  CUDF_EXPECTS(in_row_mask.type().id() == type_id::BOOL8,
               "Input row mask column must be a boolean column");

  // Update output row mask such that out_row_mask[i] = true, iff in_row_mask[i] is valid and true.
  // This is inline with the masking behavior of cudf::detail::apply_boolean_mask.
  thrust::transform(rmm::exec_policy_nosync(stream),
                    thrust::counting_iterator<cudf::size_type>(0),
                    thrust::make_counting_iterator(total_rows),
                    out_row_mask.begin<bool>() + out_row_mask_offset,
                    [is_nullable = in_row_mask.nullable(),
                     in_row_mask = in_row_mask.begin<bool>(),
                     in_bitmask  = in_row_mask.null_mask()] __device__(auto row_idx) {
                      auto const is_valid = not is_nullable or bit_is_set(in_bitmask, row_idx);
                      auto const is_true  = in_row_mask[row_idx];
                      if (is_nullable) {
                        return is_valid and is_true;
                      } else {
                        return is_true;
                      }
                    });

  // Make sure the null mask of the output row mask column is all valid after the update. This is
  // to correctly assess if a payload column data page can be pruned. An invalid row in the row mask
  // column means the corresponding data page cannot be pruned.
  if (out_row_mask.nullable()) {
    cudf::set_null_mask(out_row_mask.null_mask(), 0, total_rows, true, stream);
    out_row_mask.set_null_count(0);
  }
}

}  // namespace cudf::io::parquet::experimental::detail
