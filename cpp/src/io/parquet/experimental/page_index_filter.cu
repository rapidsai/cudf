#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2025, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "hybrid_scan_helpers.hpp"
#include "io/parquet/reader_impl_helpers.hpp"
#include "io/parquet/stats_filter_helpers.hpp"

#include <cudf/ast/detail/expression_transformer.hpp>
#include <cudf/ast/detail/operators.hpp>
#include <cudf/ast/expressions.hpp>
#include <cudf/column/column_factories.hpp>
#include <cudf/detail/iterator.cuh>
#include <cudf/detail/transform.hpp>
#include <cudf/detail/utilities/integer_utils.hpp>
#include <cudf/detail/utilities/vector_factories.hpp>
#include <cudf/io/types.hpp>
#include <cudf/utilities/error.hpp>
#include <cudf/utilities/memory_resource.hpp>
#include <cudf/utilities/span.hpp>
#include <cudf/utilities/traits.hpp>
#include <cudf/utilities/type_dispatcher.hpp>

#include <rmm/cuda_stream_view.hpp>
#include <rmm/device_buffer.hpp>
#include <rmm/exec_policy.hpp>

#include <thrust/iterator/counting_iterator.h>
#include <thrust/tabulate.h>

#include <asm-generic/errno.h>

#include <algorithm>
#include <limits>
#include <numeric>
#include <optional>
#include <unordered_set>

namespace cudf::experimental::io::parquet::detail {

namespace {

using Type = cudf::io::parquet::detail::Type;

/**
 * @brief Converts page-level statistics to 2 device columns - min, max values. Each column has
 *        number of rows equal to the total rows in all row groups.
 *
 */
struct page_stats_caster : cudf::io::parquet::detail::stats_caster_base {
  size_type total_rows;
  std::vector<cudf::io::parquet::detail::metadata> const& per_file_metadata;
  host_span<std::vector<size_type> const> row_group_indices;

  page_stats_caster(size_type total_rows,
                    std::vector<cudf::io::parquet::detail::metadata> const& per_file_metadata,
                    host_span<std::vector<size_type> const> row_group_indices)
    : total_rows{total_rows},
      per_file_metadata{per_file_metadata},
      row_group_indices{row_group_indices}
  {
  }

  // Creates device columns from column statistics (min, max)
  template <typename T>
  std::pair<std::unique_ptr<column>, std::unique_ptr<column>> operator()(
    int schema_idx,
    cudf::data_type dtype,
    rmm::cuda_stream_view stream,
    rmm::device_async_resource_ref mr) const
  {
    // List, Struct, Dictionary types are not supported
    if constexpr (cudf::is_compound<T>() && !std::is_same_v<T, string_view>) {
      CUDF_FAIL("Compound types do not have statistics");
    } else {
      host_column<T> min(total_rows, stream);
      host_column<T> max(total_rows, stream);
      size_type stats_idx = 0;

      for (size_t src_idx = 0; src_idx < row_group_indices.size(); ++src_idx) {
        for (auto const rg_idx : row_group_indices[src_idx]) {
          auto const& row_group = per_file_metadata[src_idx].row_groups[rg_idx];
          auto col              = std::find_if(row_group.columns.begin(),
                                  row_group.columns.end(),
                                  [schema_idx](cudf::io::parquet::detail::ColumnChunk const& col) {
                                    return col.schema_idx == schema_idx;
                                  });
          if (col != std::end(row_group.columns) and col->column_index.has_value()) {
            auto const& colchunk     = *col;
            auto const& column_index = colchunk.column_index.value();
            auto const& offset_index = colchunk.offset_index.value();

            CUDF_EXPECTS(column_index.min_values.size() == column_index.max_values.size(),
                         "page min and max values should be of same size");
            CUDF_EXPECTS(
              column_index.min_values.size() == offset_index.page_locations.size(),
              "mismatch between size of min/max page values and the size of page locations");
            for (size_t page_idx = 0; page_idx < column_index.min_values.size(); ++page_idx) {
              // To support deprecated min, max fields.
              auto const& min_value       = column_index.min_values[page_idx];
              auto const& max_value       = column_index.min_values[page_idx];
              int64_t const first_row_idx = offset_index.page_locations[page_idx].first_row_index;
              int64_t const last_row_idx =
                (page_idx < column_index.min_values.size() - 1)
                  ? offset_index.page_locations[page_idx + 1].first_row_index
                  : row_group.num_rows;
              for (int64_t i = first_row_idx; i < last_row_idx; ++i) {
                // translate binary data to Type then to <T>
                min.set_index(stats_idx + i, min_value, colchunk.meta_data.type);
                max.set_index(stats_idx + i, max_value, colchunk.meta_data.type);
              }
            }
          } else {
            for (int64_t i = 0; i < row_group.num_rows; ++i) {
              // Marking it null, if column present in row group
              min.set_index(stats_idx + i, std::nullopt, {});
              max.set_index(stats_idx + i, std::nullopt, {});
            }
          }
          stats_idx += row_group.num_rows;
        }
      }
      return {min.to_device(dtype, stream, mr), max.to_device(dtype, stream, mr)};
    }
  }
};

}  // namespace

std::unique_ptr<cudf::column> aggregate_reader_metadata::filter_data_pages_with_stats(
  cudf::host_span<std::vector<size_type> const> row_group_indices,
  host_span<data_type const> output_dtypes,
  host_span<int const> output_column_schemas,
  std::optional<std::reference_wrapper<ast::expression const>> filter,
  rmm::cuda_stream_view stream,
  rmm::device_async_resource_ref mr) const
{
  // Converts Column chunk statistics to a table
  // where min(col[i]) = columns[i*2], max(col[i])=columns[i*2+1]
  // For each column, it contains total number of rows from all row groups.
  size_t const total_rows = std::accumulate(
    row_group_indices.begin(),
    row_group_indices.end(),
    size_t{0},
    [&](auto sum, auto const& rg_indices) {
      return std::accumulate(
        rg_indices.begin(), rg_indices.end(), sum, [&](auto sum, auto const& rg_index) {
          CUDF_EXPECTS(
            sum + per_file_metadata[rg_index].num_rows <= std::numeric_limits<size_type>::max(),
            "Total rows exceed the maximum value");
          return sum + per_file_metadata[rg_index].num_rows;
        });
    });

  std::vector<std::unique_ptr<column>> columns;
  page_stats_caster const stats_col{
    static_cast<size_type>(total_rows), per_file_metadata, row_group_indices};
  for (size_t col_idx = 0; col_idx < output_dtypes.size(); col_idx++) {
    auto const schema_idx = output_column_schemas[col_idx];
    auto const& dtype     = output_dtypes[col_idx];
    // Only comparable types except fixed point are supported.
    if (cudf::is_compound(dtype) && dtype.id() != cudf::type_id::STRING) {
      // placeholder only for unsupported types.
      columns.push_back(cudf::make_numeric_column(
        data_type{cudf::type_id::BOOL8}, total_rows, rmm::device_buffer{}, 0, stream, mr));
      columns.push_back(cudf::make_numeric_column(
        data_type{cudf::type_id::BOOL8}, total_rows, rmm::device_buffer{}, 0, stream, mr));
      continue;
    }
    auto [min_col, max_col] =
      cudf::type_dispatcher<dispatch_storage_type>(dtype, stats_col, schema_idx, dtype, stream, mr);
    columns.push_back(std::move(min_col));
    columns.push_back(std::move(max_col));
  }
  auto stats_table = cudf::table(std::move(columns));

  // Converts AST to StatsAST with reference to min, max columns in above `stats_table`.
  cudf::io::parquet::detail::stats_expression_converter const stats_expr{
    filter.value().get(), static_cast<size_type>(output_dtypes.size())};

  // Filter the input table using AST expression
  return cudf::detail::compute_column(stats_table, stats_expr.get_stats_expr().get(), stream, mr);
}

std::vector<std::vector<cudf::io::text::byte_range_info>>
aggregate_reader_metadata::get_filter_columns_data_pages(
  cudf::column_view input_rows,
  cudf::host_span<std::vector<size_type> const> row_group_indices,
  host_span<data_type const> output_dtypes,
  host_span<int const> output_column_schemas,
  rmm::cuda_stream_view stream) const
{
  if (row_group_indices.empty()) { return {}; }

  CUDF_EXPECTS(input_rows.type().id() == cudf::type_id::BOOL8,
               "Input row bitmask should be of type BOOL8");

  auto const host_bitmask = [&] {
    auto const num_bitmasks = num_bitmask_words(input_rows.size());
    if (input_rows.nullable()) {
      return cudf::detail::make_host_vector_sync(
        device_span<bitmask_type const>(input_rows.null_mask(), num_bitmasks), stream);
    } else {
      auto bitmask = cudf::detail::make_host_vector<bitmask_type>(num_bitmasks, stream);
      std::fill(bitmask.begin(), bitmask.end(), ~bitmask_type{0});
      return bitmask;
    }
  }();

  auto validity_it = cudf::detail::make_counting_transform_iterator(
    0, [bitmask = host_bitmask.data()](auto bit_index) { return bit_is_set(bitmask, bit_index); });
  // Return only filtered row groups based on predicate
  auto const is_row_required = cudf::detail::make_host_vector_sync(
    device_span<uint8_t const>(input_rows.data<uint8_t>(), input_rows.size()), stream);

  // Return if all are required, or all are nulls.
  if (input_rows.null_count() == input_rows.size() or
      std::all_of(
        is_row_required.cbegin(), is_row_required.cend(), [](auto i) { return bool(i); })) {
    return {};
  }

  std::vector<std::vector<size_type>> page_offsets(output_dtypes.size());

  auto data_page_bytes =
    std::vector<std::vector<cudf::io::text::byte_range_info>>(output_dtypes.size());

  size_type curr_rows = 0;

  // For all columns in the output_dtypes
  for (auto schema_idx : output_column_schemas) {
    curr_rows        = 0;
    size_t num_pages = 0;
    // For all source files
    std::for_each(
      thrust::counting_iterator<size_t>(0),
      thrust::counting_iterator(row_group_indices.size()),
      [&](auto const src_index) {
        // Get all row group indices in the data source
        auto const& rg_indices = row_group_indices[src_index];
        // For all row groups in the source file
        std::for_each(rg_indices.cbegin(), rg_indices.cend(), [&](auto const rg_index) {
          auto const& row_group = per_file_metadata[src_index].row_groups[rg_index];
          auto col              = std::find_if(row_group.columns.begin(),
                                  row_group.columns.end(),
                                  [schema_idx](cudf::io::parquet::detail::ColumnChunk const& col) {
                                    return col.schema_idx == schema_idx;
                                  });
          if (col != std::end(row_group.columns) and col->column_index.has_value()) {
            auto const& colchunk     = *col;
            auto const& column_index = colchunk.column_index.value();
            auto const& offset_index = colchunk.offset_index.value();
            for (size_t page_idx = 0; page_idx < offset_index.page_locations.size(); ++page_idx) {
              page_offsets[schema_idx].push_back(
                curr_rows + offset_index.page_locations[page_idx].first_row_index);
            }
          }
          curr_rows += row_group.num_rows;
          num_pages += col->offset_index.value().page_locations.size();
        });
      });

    // Insert an empty byte range slot for each page in the column
    data_page_bytes.emplace_back(num_pages);

    // Insert the last offset of the last page
    page_offsets[schema_idx].push_back(curr_rows);
  }

  auto const total_rows = curr_rows;

  // For all rows in total rows, if (not validity_it[row_idx] or is_row_required[row_idx]), then
  // look which page it belongs to and set it to true.
  for (size_t col_idx = 0; col_idx < output_dtypes.size(); col_idx++) {
    for (size_type row_idx = 0; row_idx < total_rows; ++row_idx) {
      if (not validity_it[row_idx] or is_row_required[row_idx]) {
        // binary search to find the page index this row_idx belongs to and set the page index to
        // true page_indices
        auto const& offsets = page_offsets[col_idx];
        auto const page_itr = std::upper_bound(offsets.cbegin(), offsets.cend(), row_idx);
        CUDF_EXPECTS(page_itr != offsets.cbegin(), "Invalid page index");
        auto const page_idx = std::distance(offsets.cbegin(), page_itr) - 1;

        data_page_bytes[col_idx][page_idx] = cudf::io::text::byte_range_info{
          page_offsets[col_idx][page_idx],
          page_offsets[col_idx][page_idx + 1] - page_offsets[col_idx][page_idx]};

        // Move row_idx to the last row of the page, so that we don't need to check the same page
        // again.
        row_idx = offsets[page_idx + 1] - 1;
      }
    }
  }

  return data_page_bytes;
}

}  // namespace cudf::experimental::io::parquet::detail
