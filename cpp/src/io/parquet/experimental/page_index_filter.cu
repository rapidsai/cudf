#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2025, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "hybrid_scan_helpers.hpp"
#include "io/parquet/reader_impl_helpers.hpp"
#include "io/parquet/stats_filter_helpers.hpp"

#include <cudf/ast/detail/expression_transformer.hpp>
#include <cudf/ast/detail/operators.hpp>
#include <cudf/ast/expressions.hpp>
#include <cudf/column/column_factories.hpp>
#include <cudf/detail/iterator.cuh>
#include <cudf/detail/transform.hpp>
#include <cudf/detail/utilities/integer_utils.hpp>
#include <cudf/detail/utilities/vector_factories.hpp>
#include <cudf/io/types.hpp>
#include <cudf/utilities/error.hpp>
#include <cudf/utilities/memory_resource.hpp>
#include <cudf/utilities/span.hpp>
#include <cudf/utilities/traits.hpp>
#include <cudf/utilities/type_dispatcher.hpp>

#include <rmm/cuda_stream_view.hpp>
#include <rmm/device_buffer.hpp>
#include <rmm/exec_policy.hpp>

#include <thrust/iterator/counting_iterator.h>
#include <thrust/tabulate.h>

#include <asm-generic/errno.h>

#include <algorithm>
#include <limits>
#include <numeric>
#include <optional>
#include <unordered_set>

namespace cudf::experimental::io::parquet::detail {

namespace {

using Type = cudf::io::parquet::detail::Type;

/**
 * @brief Converts page-level statistics to 2 device columns - min, max values. Each column has
 *        number of rows equal to the total rows in all row groups.
 *
 */
struct page_stats_caster : cudf::io::parquet::detail::stats_caster_base {
  size_type total_rows;
  std::vector<cudf::io::parquet::detail::metadata> const& per_file_metadata;
  host_span<std::vector<size_type> const> row_group_indices;

  page_stats_caster(size_type total_rows,
                    std::vector<cudf::io::parquet::detail::metadata> const& per_file_metadata,
                    host_span<std::vector<size_type> const> row_group_indices)
    : total_rows{total_rows},
      per_file_metadata{per_file_metadata},
      row_group_indices{row_group_indices}
  {
  }

  // Creates device columns from column statistics (min, max)
  template <typename T>
  std::pair<std::unique_ptr<column>, std::unique_ptr<column>> operator()(
    int schema_idx,
    cudf::data_type dtype,
    rmm::cuda_stream_view stream,
    rmm::device_async_resource_ref mr) const
  {
    // List, Struct, Dictionary types are not supported
    if constexpr (cudf::is_compound<T>() && !std::is_same_v<T, string_view>) {
      CUDF_FAIL("Compound types do not have statistics");
    } else {
      // Local struct to hold host columns
      struct host_column {
        // using thrust::host_vector because std::vector<bool> uses bitmap instead of byte per bool.
        cudf::detail::host_vector<T> val;
        std::vector<bitmask_type> null_mask;
        cudf::size_type null_count = 0;
        host_column(size_type total_rows, rmm::cuda_stream_view stream)
          : val{cudf::detail::make_host_vector<T>(total_rows, stream)},
            null_mask(cudf::util::div_rounding_up_safe<cudf::size_type>(
                        cudf::bitmask_allocation_size_bytes(total_rows), sizeof(bitmask_type)),
                      ~bitmask_type{0})
        {
        }

        void set_index(size_type index,
                       std::optional<std::vector<uint8_t>> const& binary_value,
                       Type const type)
        {
          if (binary_value.has_value()) {
            val[index] = convert<T>(binary_value.value().data(), binary_value.value().size(), type);
          }
          if (not binary_value.has_value()) {
            clear_bit_unsafe(null_mask.data(), index);
            null_count++;
          }
        }

        static auto make_strings_children(host_span<string_view> host_strings,
                                          rmm::cuda_stream_view stream,
                                          rmm::device_async_resource_ref mr)
        {
          auto const total_char_count = std::accumulate(
            host_strings.begin(), host_strings.end(), 0, [](auto sum, auto const& str) {
              return sum + str.size_bytes();
            });
          auto chars = cudf::detail::make_empty_host_vector<char>(total_char_count, stream);
          auto offsets =
            cudf::detail::make_empty_host_vector<cudf::size_type>(host_strings.size() + 1, stream);
          offsets.push_back(0);
          for (auto const& str : host_strings) {
            auto tmp =
              str.empty() ? std::string_view{} : std::string_view(str.data(), str.size_bytes());
            chars.insert(chars.end(), std::cbegin(tmp), std::cend(tmp));
            offsets.push_back(offsets.back() + tmp.length());
          }
          auto d_chars   = cudf::detail::make_device_uvector_async(chars, stream, mr);
          auto d_offsets = cudf::detail::make_device_uvector_sync(offsets, stream, mr);
          return std::tuple{std::move(d_chars), std::move(d_offsets)};
        }

        auto to_device(cudf::data_type dtype,
                       rmm::cuda_stream_view stream,
                       rmm::device_async_resource_ref mr)
        {
          if constexpr (std::is_same_v<T, string_view>) {
            auto [d_chars, d_offsets] = make_strings_children(val, stream, mr);
            return cudf::make_strings_column(
              val.size(),
              std::make_unique<column>(std::move(d_offsets), rmm::device_buffer{}, 0),
              d_chars.release(),
              null_count,
              rmm::device_buffer{
                null_mask.data(), cudf::bitmask_allocation_size_bytes(val.size()), stream, mr});
          }
          return std::make_unique<column>(
            dtype,
            val.size(),
            cudf::detail::make_device_uvector_async(val, stream, mr).release(),
            rmm::device_buffer{
              null_mask.data(), cudf::bitmask_allocation_size_bytes(val.size()), stream, mr},
            null_count);
        }
      };  // local struct host_column
      host_column min(total_rows, stream);
      host_column max(total_rows, stream);
      size_type stats_idx = 0;

      for (size_t src_idx = 0; src_idx < row_group_indices.size(); ++src_idx) {
        for (auto const rg_idx : row_group_indices[src_idx]) {
          auto const& row_group = per_file_metadata[src_idx].row_groups[rg_idx];
          auto col              = std::find_if(row_group.columns.begin(),
                                  row_group.columns.end(),
                                  [schema_idx](cudf::io::parquet::detail::ColumnChunk const& col) {
                                    return col.schema_idx == schema_idx;
                                  });
          if (col != std::end(row_group.columns) and col->column_index.has_value()) {
            auto const& colchunk     = *col;
            auto const& column_index = colchunk.column_index.value();
            auto const& offset_index = colchunk.offset_index.value();

            CUDF_EXPECTS(column_index.min_values.size() == column_index.max_values.size(),
                         "page min and max values should be of same size");
            CUDF_EXPECTS(
              column_index.min_values.size() == offset_index.page_locations.size(),
              "mismatch between size of min/max page values and the size of page locations");
            for (size_t page_idx = 0; page_idx < column_index.min_values.size(); ++page_idx) {
              // To support deprecated min, max fields.
              auto const& min_value       = column_index.min_values[page_idx];
              auto const& max_value       = column_index.min_values[page_idx];
              int64_t const first_row_idx = offset_index.page_locations[page_idx].first_row_index;
              int64_t const last_row_idx =
                (page_idx < column_index.min_values.size() - 1)
                  ? offset_index.page_locations[page_idx + 1].first_row_index
                  : row_group.num_rows;
              for (int64_t i = first_row_idx; i < last_row_idx; ++i) {
                // translate binary data to Type then to <T>
                min.set_index(stats_idx + i, min_value, colchunk.meta_data.type);
                max.set_index(stats_idx + i, max_value, colchunk.meta_data.type);
              }
            }
          } else {
            for (int64_t i = 0; i < row_group.num_rows; ++i) {
              // Marking it null, if column present in row group
              min.set_index(stats_idx + i, std::nullopt, {});
              max.set_index(stats_idx + i, std::nullopt, {});
            }
          }
          stats_idx += row_group.num_rows;
        }
      }
      return {min.to_device(dtype, stream, mr), max.to_device(dtype, stream, mr)};
    }
  }
};

}  // namespace

std::unique_ptr<cudf::column> aggregate_reader_metadata::filter_data_pages_with_stats(
  cudf::host_span<std::vector<size_type> const> row_group_indices,
  host_span<data_type const> output_dtypes,
  host_span<int const> output_column_schemas,
  std::optional<std::reference_wrapper<ast::expression const>> filter,
  rmm::cuda_stream_view stream,
  rmm::device_async_resource_ref mr) const
{
  // Converts Column chunk statistics to a table
  // where min(col[i]) = columns[i*2], max(col[i])=columns[i*2+1]
  // For each column, it contains total number of rows from all row groups.
  size_t const total_rows = std::accumulate(
    row_group_indices.begin(),
    row_group_indices.end(),
    size_t{0},
    [&](auto sum, auto const& rg_indices) {
      return std::accumulate(
        rg_indices.begin(), rg_indices.end(), sum, [&](auto sum, auto const& rg_index) {
          CUDF_EXPECTS(
            sum + per_file_metadata[rg_index].num_rows <= std::numeric_limits<size_type>::max(),
            "Total rows exceed the maximum value");
          return sum + per_file_metadata[rg_index].num_rows;
        });
    });

  std::vector<std::unique_ptr<column>> columns;
  page_stats_caster const stats_col{
    static_cast<size_type>(total_rows), per_file_metadata, row_group_indices};
  for (size_t col_idx = 0; col_idx < output_dtypes.size(); col_idx++) {
    auto const schema_idx = output_column_schemas[col_idx];
    auto const& dtype     = output_dtypes[col_idx];
    // Only comparable types except fixed point are supported.
    if (cudf::is_compound(dtype) && dtype.id() != cudf::type_id::STRING) {
      // placeholder only for unsupported types.
      columns.push_back(cudf::make_numeric_column(
        data_type{cudf::type_id::BOOL8}, total_rows, rmm::device_buffer{}, 0, stream, mr));
      columns.push_back(cudf::make_numeric_column(
        data_type{cudf::type_id::BOOL8}, total_rows, rmm::device_buffer{}, 0, stream, mr));
      continue;
    }
    auto [min_col, max_col] =
      cudf::type_dispatcher<dispatch_storage_type>(dtype, stats_col, schema_idx, dtype, stream, mr);
    columns.push_back(std::move(min_col));
    columns.push_back(std::move(max_col));
  }
  auto stats_table = cudf::table(std::move(columns));

  // Converts AST to StatsAST with reference to min, max columns in above `stats_table`.
  cudf::io::parquet::detail::stats_expression_converter const stats_expr{
    filter.value().get(), static_cast<size_type>(output_dtypes.size())};

  // Filter the input table using AST expression
  return cudf::detail::compute_column(stats_table, stats_expr.get_stats_expr().get(), stream, mr);
}

std::vector<std::vector<cudf::io::text::byte_range_info>>
aggregate_reader_metadata::get_filter_columns_data_pages(
  cudf::column_view input_rows,
  cudf::host_span<std::vector<size_type> const> row_group_indices,
  host_span<data_type const> output_dtypes,
  host_span<int const> output_column_schemas,
  rmm::cuda_stream_view stream) const
{
  if (row_group_indices.empty()) { return {}; }

  CUDF_EXPECTS(input_rows.type().id() == cudf::type_id::BOOL8,
               "Input row bitmask should be of type BOOL8");

  auto const host_bitmask = [&] {
    auto const num_bitmasks = num_bitmask_words(input_rows.size());
    if (input_rows.nullable()) {
      return cudf::detail::make_host_vector_sync(
        device_span<bitmask_type const>(input_rows.null_mask(), num_bitmasks), stream);
    } else {
      auto bitmask = cudf::detail::make_host_vector<bitmask_type>(num_bitmasks, stream);
      std::fill(bitmask.begin(), bitmask.end(), ~bitmask_type{0});
      return bitmask;
    }
  }();

  auto validity_it = cudf::detail::make_counting_transform_iterator(
    0, [bitmask = host_bitmask.data()](auto bit_index) { return bit_is_set(bitmask, bit_index); });
  // Return only filtered row groups based on predicate
  auto const is_row_required = cudf::detail::make_host_vector_sync(
    device_span<uint8_t const>(input_rows.data<uint8_t>(), input_rows.size()), stream);

  // Return if all are required, or all are nulls.
  if (input_rows.null_count() == input_rows.size() or
      std::all_of(
        is_row_required.cbegin(), is_row_required.cend(), [](auto i) { return bool(i); })) {
    return {};
  }

  std::vector<std::vector<size_type>> page_offsets(output_dtypes.size());

  auto data_page_bytes =
    std::vector<std::vector<cudf::io::text::byte_range_info>>(output_dtypes.size());

  size_type curr_rows = 0;

  // For all columns in the output_dtypes
  for (auto schema_idx : output_column_schemas) {
    curr_rows        = 0;
    size_t num_pages = 0;
    // For all source files
    std::for_each(
      thrust::counting_iterator<size_t>(0),
      thrust::counting_iterator(row_group_indices.size()),
      [&](auto const src_index) {
        // Get all row group indices in the data source
        auto const& rg_indices = row_group_indices[src_index];
        // For all row groups in the source file
        std::for_each(rg_indices.cbegin(), rg_indices.cend(), [&](auto const rg_index) {
          auto const& row_group = per_file_metadata[src_index].row_groups[rg_index];
          auto col              = std::find_if(row_group.columns.begin(),
                                  row_group.columns.end(),
                                  [schema_idx](cudf::io::parquet::detail::ColumnChunk const& col) {
                                    return col.schema_idx == schema_idx;
                                  });
          if (col != std::end(row_group.columns) and col->column_index.has_value()) {
            auto const& colchunk     = *col;
            auto const& column_index = colchunk.column_index.value();
            auto const& offset_index = colchunk.offset_index.value();
            for (size_t page_idx = 0; page_idx < offset_index.page_locations.size(); ++page_idx) {
              page_offsets[schema_idx].push_back(
                curr_rows + offset_index.page_locations[page_idx].first_row_index);
            }
          }
          curr_rows += row_group.num_rows;
          num_pages += col->offset_index.value().page_locations.size();
        });
      });

    // Insert an empty byte range slot for each page in the column
    data_page_bytes.emplace_back(num_pages);

    // Insert the last offset of the last page
    page_offsets[schema_idx].push_back(curr_rows);
  }

  auto const total_rows = curr_rows;

  // For all rows in total rows, if (not validity_it[row_idx] or is_row_required[row_idx]), then
  // look which page it belongs to and set it to true.
  for (size_t col_idx = 0; col_idx < output_dtypes.size(); col_idx++) {
    for (size_type row_idx = 0; row_idx < total_rows; ++row_idx) {
      if (not validity_it[row_idx] or is_row_required[row_idx]) {
        // binary search to find the page index this row_idx belongs to and set the page index to
        // true page_indices
        auto const& offsets = page_offsets[col_idx];
        auto const page_itr = std::upper_bound(offsets.cbegin(), offsets.cend(), row_idx);
        CUDF_EXPECTS(page_itr != offsets.cbegin(), "Invalid page index");
        auto const page_idx = std::distance(offsets.cbegin(), page_itr) - 1;

        data_page_bytes[col_idx][page_idx] = cudf::io::text::byte_range_info{
          page_offsets[col_idx][page_idx],
          page_offsets[col_idx][page_idx + 1] - page_offsets[col_idx][page_idx]};

        // Move row_idx to the last row of the page, so that we don't need to check the same page
        // again.
        row_idx = offsets[page_idx + 1] - 1;
      }
    }
  }

  return data_page_bytes;
}

}  // namespace cudf::experimental::io::parquet::detail