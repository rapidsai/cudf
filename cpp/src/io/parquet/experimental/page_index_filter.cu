#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2025, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "hybrid_scan_helpers.hpp"
#include "io/parquet/reader_impl_helpers.hpp"
#include "io/parquet/stats_filter_helpers.hpp"

#include <cudf/column/column_factories.hpp>
#include <cudf/detail/iterator.cuh>
#include <cudf/detail/null_mask.hpp>
#include <cudf/detail/nvtx/ranges.hpp>
#include <cudf/detail/transform.hpp>
#include <cudf/detail/utilities/batched_memcpy.hpp>
#include <cudf/detail/utilities/host_worker_pool.hpp>
#include <cudf/detail/utilities/stream_pool.hpp>
#include <cudf/detail/utilities/vector_factories.hpp>
#include <cudf/io/types.hpp>
#include <cudf/logger.hpp>
#include <cudf/null_mask.hpp>
#include <cudf/types.hpp>
#include <cudf/utilities/error.hpp>
#include <cudf/utilities/memory_resource.hpp>
#include <cudf/utilities/span.hpp>
#include <cudf/utilities/traits.hpp>
#include <cudf/utilities/type_dispatcher.hpp>

#include <rmm/cuda_stream_view.hpp>
#include <rmm/device_buffer.hpp>
#include <rmm/device_uvector.hpp>
#include <rmm/exec_policy.hpp>

#include <thrust/gather.h>
#include <thrust/host_vector.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/logical.h>
#include <thrust/unique.h>

#include <algorithm>
#include <limits>
#include <numeric>

namespace cudf::io::parquet::experimental::detail {

using metadata_base     = parquet::detail::metadata;
using stats_caster_base = parquet::detail::stats_caster_base;
using string_index_pair = parquet::detail::string_index_pair;

namespace {

/**
 * @brief Make a device vector where each row contains the index of the page it belongs to
 */
[[nodiscard]] rmm::device_uvector<size_type> make_page_indices_async(
  cudf::host_span<cudf::size_type const> page_row_counts,
  cudf::host_span<cudf::size_type const> page_row_offsets,
  cudf::size_type total_rows,
  rmm::cuda_stream_view stream)
{
  auto mr = cudf::get_current_device_resource_ref();

  // Copy page-level row counts and offsets to device
  auto row_counts  = cudf::detail::make_device_uvector_async(page_row_counts, stream, mr);
  auto row_offsets = cudf::detail::make_device_uvector_async(page_row_offsets, stream, mr);

  // Make a zeroed device vector to store page indices of each row
  auto page_indices =
    cudf::detail::make_zeroed_device_uvector_async<cudf::size_type>(total_rows, stream, mr);

  // Scatter page indices across the their first row's index
  thrust::scatter_if(rmm::exec_policy_nosync(stream),
                     thrust::counting_iterator<size_type>(0),
                     thrust::counting_iterator<size_type>(row_counts.size()),
                     row_offsets.begin(),
                     row_counts.begin(),
                     page_indices.begin());

  // Inclusive scan with maximum to replace zeros with the (increasing) page index it belongs to.
  // Page indices are scattered at their first row's index.
  thrust::inclusive_scan(rmm::exec_policy_nosync(stream),
                         page_indices.begin(),
                         page_indices.end(),
                         page_indices.begin(),
                         cuda::maximum<cudf::size_type>());
  return page_indices;
}

/**
 * @brief Compute page row counts and page row offsets and column chunk page (count) offsets for a
 * given column schema index
 */
[[nodiscard]] auto make_page_row_counts_and_offsets(
  cudf::host_span<metadata_base const> per_file_metadata,
  cudf::host_span<std::vector<size_type> const> row_group_indices,
  size_type schema_idx,
  rmm::cuda_stream_view stream)
{
  // Compute total number of row groups
  auto const total_row_groups =
    std::accumulate(row_group_indices.begin(),
                    row_group_indices.end(),
                    size_t{0},
                    [](auto sum, auto const& rg_indices) { return sum + rg_indices.size(); });

  // Vector to store how many rows are present in each page - set initial capacity to two data pages
  // per row group
  auto page_row_counts =
    cudf::detail::make_empty_host_vector<size_type>(2 * total_row_groups, stream);
  // Vector to store the cumulative number of rows in each page - - set initial capacity to two data
  // pages per row group
  auto page_row_offsets =
    cudf::detail::make_empty_host_vector<size_type>((2 * total_row_groups) + 1, stream);
  // Vector to store the cumulative number of pages in each column chunk
  auto col_chunk_page_offsets =
    cudf::detail::make_empty_host_vector<size_type>(total_row_groups + 1, stream);

  page_row_offsets.push_back(0);
  col_chunk_page_offsets.push_back(0);

  // For all data sources
  std::for_each(
    thrust::counting_iterator<size_t>(0),
    thrust::counting_iterator(row_group_indices.size()),
    [&](auto src_idx) {
      auto const& rg_indices = row_group_indices[src_idx];
      // For all column chunks in this data source
      std::for_each(rg_indices.cbegin(), rg_indices.cend(), [&](auto rg_idx) {
        auto const& row_group = per_file_metadata[src_idx].row_groups[rg_idx];
        // Find the column chunk with the given schema index
        auto colchunk_iter = std::find_if(
          row_group.columns.begin(), row_group.columns.end(), [schema_idx](ColumnChunk const& col) {
            return col.schema_idx == schema_idx;
          });

        CUDF_EXPECTS(colchunk_iter != row_group.columns.end(),
                     "Column chunk with schema index " + std::to_string(schema_idx) +
                       " not found in row group",
                     std::invalid_argument);

        // Compute page row counts and offsets if this column chunk has column and offset indexes
        if (colchunk_iter->offset_index.has_value()) {
          CUDF_EXPECTS(colchunk_iter->column_index.has_value(),
                       "Both offset and column indexes must be present");
          // Get the offset and column indexes of the column chunk
          auto const& offset_index = colchunk_iter->offset_index.value();
          auto const& column_index = colchunk_iter->column_index.value();

          // Number of pages in this column chunk
          auto const row_group_num_pages = offset_index.page_locations.size();

          CUDF_EXPECTS(column_index.min_values.size() == column_index.max_values.size(),
                       "page min and max values should be of same size");
          CUDF_EXPECTS(column_index.min_values.size() == row_group_num_pages,
                       "mismatch between size of min/max page values and the size of page "
                       "locations");
          // Update the cumulative number of pages in this column chunk
          col_chunk_page_offsets.push_back(col_chunk_page_offsets.back() + row_group_num_pages);

          // For all pages in this column chunk, update page row counts and offsets.
          std::for_each(
            thrust::counting_iterator<size_t>(0),
            thrust::counting_iterator(row_group_num_pages),
            [&](auto const page_idx) {
              int64_t const first_row_idx = offset_index.page_locations[page_idx].first_row_index;
              // For the last page, this is simply the total number of rows in the column chunk
              int64_t const last_row_idx =
                (page_idx < row_group_num_pages - 1)
                  ? offset_index.page_locations[page_idx + 1].first_row_index
                  : row_group.num_rows;

              // Update the page row counts and offsets
              page_row_counts.push_back(last_row_idx - first_row_idx);
              page_row_offsets.push_back(page_row_offsets.back() + page_row_counts.back());
            });
        }
      });
    });

  return std::tuple{
    std::move(page_row_counts), std::move(page_row_offsets), std::move(col_chunk_page_offsets)};
}

/**
 * @brief Compute if the page index is present in all parquet data sources for all output columns
 */
[[nodiscard]] bool compute_has_page_index(
  cudf::host_span<metadata_base const> file_metadatas,
  cudf::host_span<std::vector<size_type> const> row_group_indices,
  cudf::host_span<size_type const> output_column_schemas)
{
  // For all output columns, check all parquet data sources
  return std::all_of(
    output_column_schemas.begin(), output_column_schemas.end(), [&](auto const schema_idx) {
      // For all parquet data sources
      return std::all_of(
        thrust::counting_iterator<size_t>(0),
        thrust::counting_iterator(row_group_indices.size()),
        [&](auto const src_index) {
          // For all row groups in this parquet data source
          auto const& rg_indices = row_group_indices[src_index];
          return std::all_of(rg_indices.begin(), rg_indices.end(), [&](auto const& rg_index) {
            auto const& row_group = file_metadatas[src_index].row_groups[rg_index];
            auto col              = std::find_if(
              row_group.columns.begin(),
              row_group.columns.end(),
              [schema_idx](ColumnChunk const& col) { return col.schema_idx == schema_idx; });
            // Check if the offset_index and column_index are present
            return col != file_metadatas[src_index].row_groups[rg_index].columns.end() and
                   col->offset_index.has_value() and col->column_index.has_value();
          });
        });
    });
}
/**
 * @brief Construct a vector of all required data pages from the page row counts
 */
[[nodiscard]] auto all_required_data_pages(
  cudf::host_span<cudf::detail::host_vector<size_type> const> page_row_counts)
{
  std::vector<std::vector<bool>> all_required_data_pages;
  all_required_data_pages.reserve(page_row_counts.size());
  std::transform(
    page_row_counts.begin(),
    page_row_counts.end(),
    std::back_inserter(all_required_data_pages),
    [&](auto const& col_page_counts) { return std::vector<bool>(col_page_counts.size(), true); });

  return all_required_data_pages;
};

/**
 * @brief Converts page-level statistics of a column to 2 device columns - min, max values. Each
 * column has number of rows equal to the total rows in all row groups.
 */
struct page_stats_caster : public stats_caster_base {
  cudf::size_type total_rows;
  cudf::host_span<metadata_base const> per_file_metadata;
  cudf::host_span<std::vector<size_type> const> row_group_indices;

  page_stats_caster(size_type total_rows,
                    cudf::host_span<metadata_base const> per_file_metadata,
                    cudf::host_span<std::vector<size_type> const> row_group_indices)
    : total_rows{total_rows},
      per_file_metadata{per_file_metadata},
      row_group_indices{row_group_indices}
  {
  }

  /**
   * @brief Transforms a page-level stats column to a row-level stats column for non-string types
   *
   * @tparam T The data type of the column - must be non-compound
   * @param column Mutable view of input page-level device column
   * @param page_nullmask Host nullmask of the input page-level column
   * @param page_indices Device vector containing the page index for each row index
   * @param page_row_offsets Host vector row offsets of each page
   * @param dtype The data type of the column
   * @param stream CUDA stream
   * @param mr Device memory resource
   *
   * @return A pair containing the output data buffer and nullmask
   */
  template <typename T>
  [[nodiscard]] std::pair<rmm::device_buffer, rmm::device_buffer> build_data_and_nullmask(
    mutable_column_view input_column,
    bitmask_type const* page_nullmask,
    cudf::device_span<size_type const> page_indices,
    cudf::host_span<size_type const> page_row_offsets,
    cudf::data_type dtype,
    rmm::cuda_stream_view stream,
    rmm::device_async_resource_ref mr) const
    requires(not cudf::is_compound<T>())
  {
    // Total number of pages in the column
    size_type const total_pages = page_row_offsets.size() - 1;

    // Buffer for output data
    auto output_data = rmm::device_buffer(cudf::size_of(dtype) * total_rows, stream, mr);

    // For each row index, copy over the min/max page stat value from the corresponding page.
    thrust::gather(rmm::exec_policy_nosync(stream),
                   page_indices.begin(),
                   page_indices.end(),
                   input_column.template begin<T>(),
                   reinterpret_cast<T*>(output_data.data()));

    // Buffer for output bitmask. Set all bits valid
    auto output_nullmask = cudf::create_null_mask(total_rows, mask_state::ALL_VALID, stream, mr);

    // For each input page, invalidate the null mask for corresponding rows if needed.
    std::for_each(thrust::counting_iterator(0),
                  thrust::counting_iterator(total_pages),
                  [&](auto const page_idx) {
                    if (not bit_is_set(page_nullmask, page_idx)) {
                      cudf::set_null_mask(static_cast<bitmask_type*>(output_nullmask.data()),
                                          page_row_offsets[page_idx],
                                          page_row_offsets[page_idx + 1],
                                          false,
                                          stream);
                    }
                  });

    return {std::move(output_data), std::move(output_nullmask)};
  }

  /**
   * @brief Transforms a page-level stats column to a row-level stats column for string type
   *
   * @param host_strings Host span of cudf::string_view values in the input page-level host column
   * @param host_chars Host span of string data of the input page-level host column
   * @param host_nullmask Nullmask of the input page-level host column
   * @param page_indices Device vector containing the page index for each row index
   * @param page_row_offsets Host vector row offsets of each page
   * @param stream CUDA stream
   * @param mr Device memory resource
   *
   * @return A pair containing the output data buffer and nullmask
   */
  [[nodiscard]] std::
    tuple<rmm::device_buffer, rmm::device_uvector<cudf::size_type>, rmm::device_buffer>
    build_string_data_and_nullmask(cudf::host_span<cudf::string_view const> host_strings,
                                   cudf::host_span<char const> host_chars,
                                   bitmask_type const* host_page_nullmask,
                                   cudf::device_span<size_type const> page_indices,
                                   cudf::host_span<size_type const> page_row_offsets,
                                   rmm::cuda_stream_view stream,
                                   rmm::device_async_resource_ref mr) const
  {
    // Total number of pages in the column
    size_type const total_pages = page_row_offsets.size() - 1;

    // Construct device vectors containing page-level (input) string data, and offsets and sizes
    auto [page_str_chars, page_str_offsets, page_str_sizes] =
      host_column<cudf::string_view>::make_strings_children(host_strings, host_chars, stream, mr);

    // Buffer for row-level string sizes (output).
    auto row_str_sizes = rmm::device_uvector<size_t>(total_rows, stream, mr);
    // Gather string sizes from page to row level
    thrust::gather(rmm::exec_policy_nosync(stream),
                   page_indices.begin(),
                   page_indices.end(),
                   page_str_sizes.begin(),
                   row_str_sizes.begin());

    // Total bytes in the output chars buffer
    auto const total_bytes = thrust::reduce(rmm::exec_policy(stream),
                                            row_str_sizes.begin(),
                                            row_str_sizes.end(),
                                            size_t{0},
                                            cuda::std::plus<size_t>());

    CUDF_EXPECTS(
      total_bytes <= cuda::std::numeric_limits<cudf::size_type>::max(),
      "The strings child of the page statistics column cannot exceed the column size limit");

    // page-level strings nullmask (input)
    auto const input_nullmask = host_page_nullmask;

    // Buffer for row-level strings nullmask (output). Initialize to all bits set.
    auto output_nullmask = cudf::create_null_mask(total_rows, mask_state::ALL_VALID, stream, mr);

    // For each input page, invalidate the null mask for corresponding rows if needed.
    std::for_each(thrust::counting_iterator(0),
                  thrust::counting_iterator(total_pages),
                  [&](auto const page_idx) {
                    if (not bit_is_set(input_nullmask, page_idx)) {
                      cudf::set_null_mask(static_cast<bitmask_type*>(output_nullmask.data()),
                                          page_row_offsets[page_idx],
                                          page_row_offsets[page_idx + 1],
                                          false,
                                          stream);
                    }
                  });

    // Buffer for row-level string offsets (output).
    auto row_str_offsets =
      cudf::detail::make_zeroed_device_uvector_async<cudf::size_type>(total_rows + 1, stream, mr);
    thrust::inclusive_scan(rmm::exec_policy_nosync(stream),
                           row_str_sizes.begin(),
                           row_str_sizes.end(),
                           row_str_offsets.begin() + 1);

    // Buffer for row-level string chars (output).
    auto row_str_chars = rmm::device_buffer(total_bytes, stream, mr);

    // Iterator for input (page-level) string chars
    auto src_iter = thrust::make_transform_iterator(
      thrust::make_counting_iterator<size_t>(0),
      cuda::proclaim_return_type<char*>(
        [chars        = page_str_chars.begin(),
         offsets      = page_str_offsets.begin(),
         page_indices = page_indices.begin()] __device__(size_t index) {
          auto const page_index = page_indices[index];
          return chars + offsets[page_index];
        }));

    // Iterator for output (row-level) string chars
    auto dst_iter = thrust::make_transform_iterator(
      thrust::make_counting_iterator<size_t>(0),
      cuda::proclaim_return_type<char*>(
        [chars   = reinterpret_cast<char*>(row_str_chars.data()),
         offsets = row_str_offsets.begin()] __device__(size_t index) {
          return chars + offsets[index];
        }));

    // Iterator for string sizes
    auto size_iter = thrust::make_transform_iterator(
      thrust::make_counting_iterator<size_t>(0),
      cuda::proclaim_return_type<size_t>(
        [sizes = row_str_sizes.begin()] __device__(size_t index) { return sizes[index]; }));

    // Gather page-level string chars to row-level string chars
    cudf::detail::batched_memcpy_async(src_iter, dst_iter, size_iter, total_rows, stream);

    // Return row-level (output) strings children and the nullmask
    return std::tuple{
      std::move(row_str_chars), std::move(row_str_offsets), std::move(output_nullmask)};
  }

  /**
   * @brief Builds two device columns storing the corresponding page-level statistics (min, max)
   *        respectively of a column at each row index.
   *
   * @tparam T underlying type of the column
   * @param schema_idx Column schema index
   * @param dtype Column data type
   * @param stream CUDA stream
   * @param mr Device memory resource
   *
   * @return A pair of device columns with min and max value from page statistics for each row
   */
  template <typename T>
  std::pair<std::unique_ptr<column>, std::unique_ptr<column>> operator()(
    cudf::size_type schema_idx,
    cudf::data_type dtype,
    rmm::cuda_stream_view stream,
    rmm::device_async_resource_ref mr) const
  {
    // List, Struct, Dictionary types are not supported
    if constexpr (cudf::is_compound<T>() and not cuda::std::is_same_v<T, string_view>) {
      CUDF_FAIL("Compound types other than strings do not have statistics");
    } else {
      // Compute column chunk level page count offsets, and page level row counts and row offsets.
      auto const [page_row_counts, page_row_offsets, col_chunk_page_offsets] =
        make_page_row_counts_and_offsets(per_file_metadata, row_group_indices, schema_idx, stream);

      CUDF_EXPECTS(
        page_row_offsets.back() == total_rows,
        "The number of rows must be equal across row groups and pages within row groups");

      auto const total_pages = col_chunk_page_offsets.back();

      // Create host columns with page-level min, max values
      host_column<T> min(total_pages, stream);
      host_column<T> max(total_pages, stream);

      // Populate the host columns with page-level min, max statistics from the page index
      auto page_offset_idx = 0;
      // For all row data sources
      std::for_each(
        thrust::counting_iterator<size_t>(0),
        thrust::counting_iterator(row_group_indices.size()),
        [&](auto src_idx) {
          // For all column chunks in this source
          auto const& rg_indices = row_group_indices[src_idx];
          std::for_each(rg_indices.cbegin(), rg_indices.cend(), [&](auto rg_idx) {
            auto const& row_group = per_file_metadata[src_idx].row_groups[rg_idx];
            // Find colchunk_iter in row_group.columns. Guaranteed to be found as already verified
            // in make_page_row_counts_and_offsets()
            auto colchunk_iter = std::find_if(
              row_group.columns.begin(),
              row_group.columns.end(),
              [schema_idx](ColumnChunk const& col) { return col.schema_idx == schema_idx; });

            auto const& colchunk               = *colchunk_iter;
            auto const& column_index           = colchunk.column_index.value();
            auto const num_pages_in_colchunk   = column_index.min_values.size();
            auto const page_offset_in_colchunk = col_chunk_page_offsets[page_offset_idx++];

            // For all pages in this column chunk
            std::for_each(
              thrust::counting_iterator<size_t>(0),
              thrust::counting_iterator(num_pages_in_colchunk),
              [&](auto page_idx) {
                auto const& min_value = column_index.min_values[page_idx];
                auto const& max_value = column_index.max_values[page_idx];
                // Translate binary data to Type then to <T>
                min.set_index(
                  page_offset_in_colchunk + page_idx, min_value, colchunk.meta_data.type);
                max.set_index(
                  page_offset_in_colchunk + page_idx, max_value, colchunk.meta_data.type);
              });
          });
        });

      // Construct a row indices mapping based on page row counts and offsets
      auto const page_indices =
        make_page_indices_async(page_row_counts, page_row_offsets, total_rows, stream);

      // For non-strings columns, directly gather the page-level column data and bitmask to the
      // row-level.
      if constexpr (not cuda::std::is_same_v<T, cudf::string_view>) {
        // Move host columns to device
        auto mincol = min.to_device(dtype, stream, mr);
        auto maxcol = max.to_device(dtype, stream, mr);

        // Convert page-level min and max columns to row-level min and max columns by gathering
        // values based on page-level row offsets
        auto [min_data, min_bitmask] = build_data_and_nullmask<T>(mincol->mutable_view(),
                                                                  min.null_mask.data(),
                                                                  page_indices,
                                                                  page_row_offsets,
                                                                  dtype,
                                                                  stream,
                                                                  mr);
        auto [max_data, max_bitmask] = build_data_and_nullmask<T>(maxcol->mutable_view(),
                                                                  max.null_mask.data(),
                                                                  page_indices,
                                                                  page_row_offsets,
                                                                  dtype,
                                                                  stream,
                                                                  mr);

        // Count nulls in min and max columns
        auto const min_nulls = cudf::detail::null_count(
          reinterpret_cast<bitmask_type*>(min_bitmask.data()), 0, total_rows, stream);
        auto const max_nulls = cudf::detail::null_count(
          reinterpret_cast<bitmask_type*>(max_bitmask.data()), 0, total_rows, stream);

        // Return min and max device columns
        return {std::make_unique<column>(
                  dtype, total_rows, std::move(min_data), std::move(min_bitmask), min_nulls),
                std::make_unique<column>(
                  dtype, total_rows, std::move(max_data), std::move(max_bitmask), max_nulls)};
      }
      // For strings columns, gather the page-level string offsets and bitmask to row-level
      // directly and gather string chars using a batched memcpy.
      else {
        auto [min_data, min_offsets, min_nullmask] = build_string_data_and_nullmask(
          min.val, min.chars, min.null_mask.data(), page_indices, page_row_offsets, stream, mr);
        auto [max_data, max_offsets, max_nullmask] = build_string_data_and_nullmask(
          max.val, max.chars, max.null_mask.data(), page_indices, page_row_offsets, stream, mr);

        // Count nulls in min and max columns
        auto const min_nulls = cudf::detail::null_count(
          reinterpret_cast<bitmask_type*>(min_nullmask.data()), 0, total_rows, stream);
        auto const max_nulls = cudf::detail::null_count(
          reinterpret_cast<bitmask_type*>(max_nullmask.data()), 0, total_rows, stream);

        // Return min and max device strings columns
        return {
          cudf::make_strings_column(
            total_rows,
            std::make_unique<column>(std::move(min_offsets), rmm::device_buffer{0, stream, mr}, 0),
            std::move(min_data),
            min_nulls,
            std::move(min_nullmask)),
          cudf::make_strings_column(
            total_rows,
            std::make_unique<column>(std::move(max_offsets), rmm::device_buffer{0, stream, mr}, 0),
            std::move(max_data),
            max_nulls,
            std::move(max_nullmask))};
      }
    }
  }
};

/**
 * @brief Functor to compute if a row in the row mask is required
 *
 * The row is required if the row mask value at row_index is either invalid or a valid `true`
 *
 * @param is_nullable Whether the row mask is nullable
 * @param nullmask The nullmask of the row mask
 * @param row_mask_data The row mask data values
 *
 * @return True if the row is valid, false otherwise.
 */
struct is_row_required_fn {
  bool is_nullable;
  bitmask_type const* nullmask;
  bool const* row_mask_data;

  __device__ bool operator()(size_type row_index) const
  {
    auto const is_invalid = is_nullable and not bit_is_set(nullmask, row_index);
    return is_invalid or row_mask_data[row_index];
  }
};

}  // namespace

std::unique_ptr<cudf::column> aggregate_reader_metadata::build_row_mask_with_page_index_stats(
  cudf::host_span<std::vector<size_type> const> row_group_indices,
  cudf::host_span<cudf::data_type const> output_dtypes,
  cudf::host_span<cudf::size_type const> output_column_schemas,
  std::reference_wrapper<ast::expression const> filter,
  rmm::cuda_stream_view stream,
  rmm::device_async_resource_ref mr) const
{
  CUDF_FUNC_RANGE();

  // Return if empty row group indices
  if (row_group_indices.empty()) { return cudf::make_empty_column(cudf::type_id::BOOL8); }

  // Check if we have page index for all columns in all row groups
  auto const has_page_index =
    compute_has_page_index(per_file_metadata, row_group_indices, output_column_schemas);

  // Return if page index is not present
  CUDF_EXPECTS(has_page_index,
               "Page pruning requires the Parquet page index for all output columns",
               std::runtime_error);

  // Total number of rows
  auto const total_rows = std::accumulate(
    thrust::counting_iterator<size_t>(0),
    thrust::counting_iterator(row_group_indices.size()),
    size_t{0},
    [&](auto sum, auto const src_index) {
      auto const& rg_indices = row_group_indices[src_index];
      return std::accumulate(
        rg_indices.begin(), rg_indices.end(), sum, [&](auto subsum, auto const rg_index) {
          CUDF_EXPECTS(subsum + per_file_metadata[src_index].row_groups[rg_index].num_rows <=
                         std::numeric_limits<size_type>::max(),
                       "Total rows exceed the maximum value");
          return subsum + per_file_metadata[src_index].row_groups[rg_index].num_rows;
        });
    });

  auto const num_columns = output_dtypes.size();

  // Get a boolean mask indicating which columns will participate in stats based filtering
  auto const stats_columns_mask =
    parquet::detail::stats_columns_collector{filter.get(),
                                             static_cast<size_type>(output_dtypes.size())}
      .get_stats_columns_mask();

  // Convert page statistics to a table
  // where min(col[i]) = columns[i*2], max(col[i])=columns[i*2+1]
  // For each column, it contains total number of rows from all row groups.
  page_stats_caster const stats_col{
    static_cast<size_type>(total_rows), per_file_metadata, row_group_indices};

  std::vector<std::unique_ptr<column>> columns;
  std::for_each(
    thrust::counting_iterator<size_t>(0),
    thrust::counting_iterator(num_columns),
    [&](auto col_idx) {
      auto const schema_idx = output_column_schemas[col_idx];
      auto const& dtype     = output_dtypes[col_idx];
      // Only participating columns and comparable types except fixed point are supported
      if (not stats_columns_mask[col_idx] or
          (cudf::is_compound(dtype) && dtype.id() != cudf::type_id::STRING)) {
        // Placeholder for unsupported types and non-participating columns
        columns.push_back(cudf::make_numeric_column(
          data_type{cudf::type_id::BOOL8}, total_rows, rmm::device_buffer{}, 0, stream, mr));
        columns.push_back(cudf::make_numeric_column(
          data_type{cudf::type_id::BOOL8}, total_rows, rmm::device_buffer{}, 0, stream, mr));
        return;
      }
      auto [min_col, max_col] = cudf::type_dispatcher<dispatch_storage_type>(
        dtype, stats_col, schema_idx, dtype, stream, mr);
      columns.push_back(std::move(min_col));
      columns.push_back(std::move(max_col));
    });

  auto stats_table = cudf::table(std::move(columns));

  // Converts AST to StatsAST with reference to min, max columns in above `stats_table`.
  parquet::detail::stats_expression_converter const stats_expr{
    filter.get(), static_cast<size_type>(output_dtypes.size())};

  // Filter the input table using AST expression and return the (BOOL8) predicate column.
  return cudf::detail::compute_column(stats_table, stats_expr.get_stats_expr().get(), stream, mr);
}

std::vector<std::vector<bool>> aggregate_reader_metadata::compute_data_page_mask(
  cudf::column_view row_mask,
  cudf::host_span<std::vector<size_type> const> row_group_indices,
  cudf::host_span<cudf::data_type const> output_dtypes,
  cudf::host_span<cudf::size_type const> output_column_schemas,
  rmm::cuda_stream_view stream) const
{
  CUDF_EXPECTS(row_mask.type().id() == cudf::type_id::BOOL8,
               "Input row bitmask should be of type BOOL8");

  auto const total_rows  = row_mask.size();
  auto const num_columns = output_dtypes.size();

  auto const has_page_index =
    compute_has_page_index(per_file_metadata, row_group_indices, output_column_schemas);

  // TODO: Don't use page pruning in case of lists and structs until we support them
  if (not has_page_index) {
    CUDF_LOG_WARN("Encountered missing Parquet page index for one or more output columns");
    return {};  // An empty data page mask indicates all pages are required
  }

  // Compute page row counts, offsets, and column chunk page offsets for each column
  std::vector<cudf::detail::host_vector<size_type>> page_row_counts;
  std::vector<cudf::detail::host_vector<size_type>> page_row_offsets;
  std::vector<cudf::detail::host_vector<size_type>> col_chunk_page_offsets;
  page_row_counts.reserve(num_columns);
  page_row_offsets.reserve(num_columns);
  col_chunk_page_offsets.reserve(num_columns);

  if (num_columns == 1) {
    auto const schema_idx = output_column_schemas[0];
    auto [counts, offsets, chunk_offsets] =
      make_page_row_counts_and_offsets(per_file_metadata, row_group_indices, schema_idx, stream);
    page_row_counts.emplace_back(std::move(counts));
    page_row_offsets.emplace_back(std::move(offsets));
  } else {
    std::vector<std::future<std::tuple<cudf::detail::host_vector<size_type>,
                                       cudf::detail::host_vector<size_type>,
                                       cudf::detail::host_vector<size_type>>>>
      page_row_counts_and_offsets_tasks;
    page_row_counts_and_offsets_tasks.reserve(num_columns);

    auto streams = cudf::detail::fork_streams(stream, num_columns);

    std::for_each(thrust::counting_iterator<size_t>(0),
                  thrust::counting_iterator(num_columns),
                  [&](auto const col_idx) {
                    page_row_counts_and_offsets_tasks.emplace_back(
                      cudf::detail::host_worker_pool().submit_task([&, col_idx = col_idx] {
                        auto const schema_idx = output_column_schemas[col_idx];
                        return make_page_row_counts_and_offsets(
                          per_file_metadata, row_group_indices, schema_idx, streams[col_idx]);
                      }));
                  });

    // Collect results from all tasks
    std::for_each(page_row_counts_and_offsets_tasks.begin(),
                  page_row_counts_and_offsets_tasks.end(),
                  [&](auto& task) {
                    auto [counts, offsets, chunk_offsets] = std::move(task).get();
                    page_row_counts.emplace_back(std::move(counts));
                    page_row_offsets.emplace_back(std::move(offsets));
                    col_chunk_page_offsets.emplace_back(std::move(chunk_offsets));
                  });
  }

  CUDF_EXPECTS(page_row_offsets.back().back() == total_rows,
               "Mismatch in total rows in input row mask and row groups",
               std::invalid_argument);

  // Return if all rows are required or all are invalid.
  if (row_mask.null_count() == row_mask.size() or thrust::all_of(rmm::exec_policy(stream),
                                                                 row_mask.begin<bool>(),
                                                                 row_mask.end<bool>(),
                                                                 cuda::std::identity{})) {
    return all_required_data_pages(page_row_counts);
  }

  auto const mr = cudf::get_current_device_resource_ref();

  // Vector to hold data page mask for each column
  auto data_page_mask = std::vector<std::vector<bool>>();
  data_page_mask.reserve(num_columns);

  auto total_surviving_pages = size_t{0};

  // For all columns, look up which pages contain at least one required row. i.e.
  // !validity_it[row_idx] or is_row_required[row_idx] satisfies, and add its byte range to the
  // output list of byte ranges for the column.
  std::for_each(
    thrust::counting_iterator<size_t>(0),
    thrust::counting_iterator(num_columns),
    [&](auto const col_idx) {
      // Construct a row indices mapping based on page row counts and offsets
      auto const total_pages_in_this_column = page_row_counts[col_idx].size();

      auto const page_indices = make_page_indices_async(
        page_row_counts[col_idx], page_row_offsets[col_idx], total_rows, stream);

      // Device vector to hold page indices with at least one required row
      rmm::device_uvector<size_type> select_page_indices(total_rows, stream, mr);

      // Copy page indices with at least one required row
      auto const filtered_pages_end_iter = thrust::copy_if(
        rmm::exec_policy_nosync(stream),
        page_indices.begin(),
        page_indices.end(),
        thrust::counting_iterator<size_type>(0),
        select_page_indices.begin(),
        is_row_required_fn{row_mask.nullable(), row_mask.null_mask(), row_mask.data<bool>()});

      // Remove duplicate page indices across (presorted) rows
      auto const filtered_uniq_page_end_iter = thrust::unique(
        rmm::exec_policy_nosync(stream), select_page_indices.begin(), filtered_pages_end_iter);

      // Number of final filtered pages for this column
      size_t const num_surviving_pages_this_column =
        thrust::distance(select_page_indices.begin(), filtered_uniq_page_end_iter);

      total_surviving_pages += num_surviving_pages_this_column;

      // Copy the filtered page indices for this column to host
      auto host_select_page_indices = cudf::detail::make_host_vector(
        cudf::device_span<cudf::size_type const>{select_page_indices.data(),
                                                 num_surviving_pages_this_column},
        stream);

      // Vector to data page mask the this column
      auto valid_pages = std::vector<bool>(total_pages_in_this_column, false);
      std::for_each(host_select_page_indices.begin(),
                    host_select_page_indices.end(),
                    [&](auto const page_idx) { valid_pages[page_idx] = true; });

      data_page_mask.push_back(std::move(valid_pages));
    });

  // Total number of input pages across all columns
  auto const total_pages = std::accumulate(
    page_row_counts.cbegin(),
    page_row_counts.cend(),
    size_t{0},
    [](auto sum, auto const& page_row_counts) { return sum + page_row_counts.size(); });

  CUDF_EXPECTS(
    total_surviving_pages <= total_pages,
    "Number of surviving pages must be less than or equal to the total number of input pages");

  return data_page_mask;
}

}  // namespace cudf::io::parquet::experimental::detail
