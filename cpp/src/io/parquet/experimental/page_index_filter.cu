#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2025, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "hybrid_scan_helpers.hpp"
#include "io/parquet/stats_filter_helpers.hpp"
#include "page_index_filter_utils.hpp"

#include <cudf/column/column_factories.hpp>
#include <cudf/detail/iterator.cuh>
#include <cudf/detail/null_mask.hpp>
#include <cudf/detail/nvtx/ranges.hpp>
#include <cudf/detail/transform.hpp>
#include <cudf/detail/utilities/batched_memcpy.hpp>
#include <cudf/detail/utilities/grid_1d.cuh>
#include <cudf/detail/utilities/host_worker_pool.hpp>
#include <cudf/detail/utilities/integer_utils.hpp>
#include <cudf/detail/utilities/stream_pool.hpp>
#include <cudf/detail/utilities/vector_factories.hpp>
#include <cudf/logger.hpp>
#include <cudf/null_mask.hpp>
#include <cudf/types.hpp>
#include <cudf/utilities/error.hpp>
#include <cudf/utilities/span.hpp>
#include <cudf/utilities/traits.hpp>
#include <cudf/utilities/type_dispatcher.hpp>

#include <rmm/cuda_stream_view.hpp>
#include <rmm/device_buffer.hpp>
#include <rmm/device_uvector.hpp>
#include <rmm/exec_policy.hpp>

#include <thrust/gather.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/logical.h>

#include <algorithm>
#include <limits>

namespace cudf::io::parquet::experimental::detail {

using metadata_base     = parquet::detail::metadata;
using stats_caster_base = parquet::detail::stats_caster_base;
using string_index_pair = parquet::detail::string_index_pair;

namespace {

/**
 * @brief Converts page-level statistics of a column to 2 device columns - min, max values. Each
 * column has number of rows equal to the total rows in all row groups.
 */
struct page_stats_caster : public stats_caster_base {
  cudf::size_type total_rows;
  cudf::host_span<metadata_base const> per_file_metadata;
  cudf::host_span<std::vector<size_type> const> row_group_indices;

  page_stats_caster(size_type total_rows,
                    cudf::host_span<metadata_base const> per_file_metadata,
                    cudf::host_span<std::vector<size_type> const> row_group_indices)
    : total_rows{total_rows},
      per_file_metadata{per_file_metadata},
      row_group_indices{row_group_indices}
  {
  }

  /**
   * @brief Transforms a page-level stats column to a row-level stats column for non-string types
   *
   * @tparam T The data type of the column - must be non-compound
   * @param column Mutable view of input page-level device column
   * @param page_nullmask Host nullmask of the input page-level column
   * @param page_indices Device vector containing the page index for each row index
   * @param page_row_offsets Host vector row offsets of each page
   * @param dtype The data type of the column
   * @param stream CUDA stream
   * @param mr Device memory resource
   *
   * @return A pair containing the output data buffer and nullmask
   */
  template <typename T>
  [[nodiscard]] std::pair<rmm::device_buffer, rmm::device_buffer> build_data_and_nullmask(
    mutable_column_view input_column,
    bitmask_type const* page_nullmask,
    cudf::device_span<size_type const> page_indices,
    cudf::host_span<size_type const> page_row_offsets,
    cudf::data_type dtype,
    rmm::cuda_stream_view stream,
    rmm::device_async_resource_ref mr) const
    requires(not cudf::is_compound<T>())
  {
    // Total number of pages in the column
    size_type const total_pages = page_row_offsets.size() - 1;

    // Buffer for output data
    auto output_data = rmm::device_buffer(cudf::size_of(dtype) * total_rows, stream, mr);

    // For each row index, copy over the min/max page stat value from the corresponding page.
    thrust::gather(rmm::exec_policy_nosync(stream),
                   page_indices.begin(),
                   page_indices.end(),
                   input_column.template begin<T>(),
                   reinterpret_cast<T*>(output_data.data()));

    // Buffer for output bitmask. Set all bits valid
    auto output_nullmask = cudf::create_null_mask(total_rows, mask_state::ALL_VALID, stream, mr);

    // For each input page, invalidate the null mask for corresponding rows if needed.
    std::for_each(thrust::counting_iterator(0),
                  thrust::counting_iterator(total_pages),
                  [&](auto const page_idx) {
                    if (not bit_is_set(page_nullmask, page_idx)) {
                      cudf::set_null_mask(static_cast<bitmask_type*>(output_nullmask.data()),
                                          page_row_offsets[page_idx],
                                          page_row_offsets[page_idx + 1],
                                          false,
                                          stream);
                    }
                  });

    return {std::move(output_data), std::move(output_nullmask)};
  }

  /**
   * @brief Transforms a page-level stats column to a row-level stats column for string type
   *
   * @param host_strings Host span of cudf::string_view values in the input page-level host column
   * @param host_chars Host span of string data of the input page-level host column
   * @param host_nullmask Nullmask of the input page-level host column
   * @param page_indices Device vector containing the page index for each row index
   * @param page_row_offsets Host vector row offsets of each page
   * @param stream CUDA stream
   * @param mr Device memory resource
   *
   * @return A pair containing the output data buffer and nullmask
   */
  [[nodiscard]] std::
    tuple<rmm::device_buffer, rmm::device_uvector<cudf::size_type>, rmm::device_buffer>
    build_string_data_and_nullmask(cudf::host_span<cudf::string_view const> host_strings,
                                   cudf::host_span<char const> host_chars,
                                   bitmask_type const* host_page_nullmask,
                                   cudf::device_span<size_type const> page_indices,
                                   cudf::host_span<size_type const> page_row_offsets,
                                   rmm::cuda_stream_view stream,
                                   rmm::device_async_resource_ref mr) const
  {
    // Total number of pages in the column
    size_type const total_pages = page_row_offsets.size() - 1;

    // Construct device vectors containing page-level (input) string data, and offsets and sizes
    auto [page_str_chars, page_str_offsets, page_str_sizes] =
      host_column<cudf::string_view>::make_strings_children(host_strings, host_chars, stream, mr);

    // Buffer for row-level string sizes (output).
    auto row_str_sizes = rmm::device_uvector<size_t>(total_rows, stream, mr);
    // Gather string sizes from page to row level
    thrust::gather(rmm::exec_policy_nosync(stream),
                   page_indices.begin(),
                   page_indices.end(),
                   page_str_sizes.begin(),
                   row_str_sizes.begin());

    // Total bytes in the output chars buffer
    auto const total_bytes = thrust::reduce(rmm::exec_policy(stream),
                                            row_str_sizes.begin(),
                                            row_str_sizes.end(),
                                            size_t{0},
                                            cuda::std::plus<size_t>());

    CUDF_EXPECTS(
      total_bytes <= cuda::std::numeric_limits<cudf::size_type>::max(),
      "The strings child of the page statistics column cannot exceed the column size limit");

    // page-level strings nullmask (input)
    auto const input_nullmask = host_page_nullmask;

    // Buffer for row-level strings nullmask (output). Initialize to all bits set.
    auto output_nullmask = cudf::create_null_mask(total_rows, mask_state::ALL_VALID, stream, mr);

    // For each input page, invalidate the null mask for corresponding rows if needed.
    std::for_each(thrust::counting_iterator(0),
                  thrust::counting_iterator(total_pages),
                  [&](auto const page_idx) {
                    if (not bit_is_set(input_nullmask, page_idx)) {
                      cudf::set_null_mask(static_cast<bitmask_type*>(output_nullmask.data()),
                                          page_row_offsets[page_idx],
                                          page_row_offsets[page_idx + 1],
                                          false,
                                          stream);
                    }
                  });

    // Buffer for row-level string offsets (output).
    auto row_str_offsets =
      cudf::detail::make_zeroed_device_uvector_async<cudf::size_type>(total_rows + 1, stream, mr);
    thrust::inclusive_scan(rmm::exec_policy_nosync(stream),
                           row_str_sizes.begin(),
                           row_str_sizes.end(),
                           row_str_offsets.begin() + 1);

    // Buffer for row-level string chars (output).
    auto row_str_chars = rmm::device_buffer(total_bytes, stream, mr);

    // Iterator for input (page-level) string chars
    auto src_iter = thrust::make_transform_iterator(
      thrust::make_counting_iterator<size_t>(0),
      cuda::proclaim_return_type<char*>(
        [chars        = page_str_chars.begin(),
         offsets      = page_str_offsets.begin(),
         page_indices = page_indices.begin()] __device__(size_t index) {
          auto const page_index = page_indices[index];
          return chars + offsets[page_index];
        }));

    // Iterator for output (row-level) string chars
    auto dst_iter = thrust::make_transform_iterator(
      thrust::make_counting_iterator<size_t>(0),
      cuda::proclaim_return_type<char*>(
        [chars   = reinterpret_cast<char*>(row_str_chars.data()),
         offsets = row_str_offsets.begin()] __device__(size_t index) {
          return chars + offsets[index];
        }));

    // Iterator for string sizes
    auto size_iter = thrust::make_transform_iterator(
      thrust::make_counting_iterator<size_t>(0),
      cuda::proclaim_return_type<size_t>(
        [sizes = row_str_sizes.begin()] __device__(size_t index) { return sizes[index]; }));

    // Gather page-level string chars to row-level string chars
    cudf::detail::batched_memcpy_async(src_iter, dst_iter, size_iter, total_rows, stream);

    // Return row-level (output) strings children and the nullmask
    return std::tuple{
      std::move(row_str_chars), std::move(row_str_offsets), std::move(output_nullmask)};
  }

  /**
   * @brief Builds two device columns storing the corresponding page-level statistics (min, max)
   *        respectively of a column at each row index.
   *
   * @tparam T underlying type of the column
   * @param schema_idx Column schema index
   * @param dtype Column data type
   * @param stream CUDA stream
   * @param mr Device memory resource
   *
   * @return A pair of device columns with min and max value from page statistics for each row
   */
  template <typename T>
  std::pair<std::unique_ptr<column>, std::unique_ptr<column>> operator()(
    cudf::size_type schema_idx,
    cudf::data_type dtype,
    rmm::cuda_stream_view stream,
    rmm::device_async_resource_ref mr) const
  {
    // List, Struct, Dictionary types are not supported
    if constexpr (cudf::is_compound<T>() and not cuda::std::is_same_v<T, string_view>) {
      CUDF_FAIL("Compound types other than strings do not have statistics");
    } else {
      // Compute column chunk level page count offsets, and page level row counts and row offsets.
      auto const [page_row_counts, page_row_offsets, col_chunk_page_offsets] =
        compute_page_row_counts_and_offsets(
          per_file_metadata, row_group_indices, schema_idx, stream);

      CUDF_EXPECTS(
        page_row_offsets.back() == total_rows,
        "The number of rows must be equal across row groups and pages within row groups");

      auto const total_pages = col_chunk_page_offsets.back();

      // Create host columns with page-level min, max values
      host_column<T> min(total_pages, stream);
      host_column<T> max(total_pages, stream);

      // Populate the host columns with page-level min, max statistics from the page index
      auto page_offset_idx = 0;
      // For all row data sources
      std::for_each(
        thrust::counting_iterator<size_t>(0),
        thrust::counting_iterator(row_group_indices.size()),
        [&](auto src_idx) {
          // For all column chunks in this source
          auto const& rg_indices = row_group_indices[src_idx];
          std::for_each(rg_indices.cbegin(), rg_indices.cend(), [&](auto rg_idx) {
            auto const& row_group = per_file_metadata[src_idx].row_groups[rg_idx];
            // Find colchunk_iter in row_group.columns. Guaranteed to be found as already verified
            // in compute_page_row_counts_and_offsets()
            auto colchunk_iter = std::find_if(
              row_group.columns.begin(),
              row_group.columns.end(),
              [schema_idx](ColumnChunk const& col) { return col.schema_idx == schema_idx; });

            auto const& colchunk               = *colchunk_iter;
            auto const& column_index           = colchunk.column_index.value();
            auto const num_pages_in_colchunk   = column_index.min_values.size();
            auto const page_offset_in_colchunk = col_chunk_page_offsets[page_offset_idx++];

            // For all pages in this column chunk
            std::for_each(
              thrust::counting_iterator<size_t>(0),
              thrust::counting_iterator(num_pages_in_colchunk),
              [&](auto page_idx) {
                auto const& min_value = column_index.min_values[page_idx];
                auto const& max_value = column_index.max_values[page_idx];
                // Translate binary data to Type then to <T>
                min.set_index(
                  page_offset_in_colchunk + page_idx, min_value, colchunk.meta_data.type);
                max.set_index(
                  page_offset_in_colchunk + page_idx, max_value, colchunk.meta_data.type);
              });
          });
        });

      // Construct a row indices mapping based on page row counts and offsets
      auto const page_indices =
        make_page_indices_async(page_row_counts, page_row_offsets, total_rows, stream);

      // For non-strings columns, directly gather the page-level column data and bitmask to the
      // row-level.
      if constexpr (not cuda::std::is_same_v<T, cudf::string_view>) {
        // Move host columns to device
        auto mincol = min.to_device(dtype, stream, mr);
        auto maxcol = max.to_device(dtype, stream, mr);

        // Convert page-level min and max columns to row-level min and max columns by gathering
        // values based on page-level row offsets
        auto [min_data, min_bitmask] = build_data_and_nullmask<T>(mincol->mutable_view(),
                                                                  min.null_mask.data(),
                                                                  page_indices,
                                                                  page_row_offsets,
                                                                  dtype,
                                                                  stream,
                                                                  mr);
        auto [max_data, max_bitmask] = build_data_and_nullmask<T>(maxcol->mutable_view(),
                                                                  max.null_mask.data(),
                                                                  page_indices,
                                                                  page_row_offsets,
                                                                  dtype,
                                                                  stream,
                                                                  mr);

        // Count nulls in min and max columns
        auto const min_nulls = cudf::detail::null_count(
          reinterpret_cast<bitmask_type*>(min_bitmask.data()), 0, total_rows, stream);
        auto const max_nulls = cudf::detail::null_count(
          reinterpret_cast<bitmask_type*>(max_bitmask.data()), 0, total_rows, stream);

        // Return min and max device columns
        return {std::make_unique<column>(
                  dtype, total_rows, std::move(min_data), std::move(min_bitmask), min_nulls),
                std::make_unique<column>(
                  dtype, total_rows, std::move(max_data), std::move(max_bitmask), max_nulls)};
      }
      // For strings columns, gather the page-level string offsets and bitmask to row-level
      // directly and gather string chars using a batched memcpy.
      else {
        auto [min_data, min_offsets, min_nullmask] = build_string_data_and_nullmask(
          min.val, min.chars, min.null_mask.data(), page_indices, page_row_offsets, stream, mr);
        auto [max_data, max_offsets, max_nullmask] = build_string_data_and_nullmask(
          max.val, max.chars, max.null_mask.data(), page_indices, page_row_offsets, stream, mr);

        // Count nulls in min and max columns
        auto const min_nulls = cudf::detail::null_count(
          reinterpret_cast<bitmask_type*>(min_nullmask.data()), 0, total_rows, stream);
        auto const max_nulls = cudf::detail::null_count(
          reinterpret_cast<bitmask_type*>(max_nullmask.data()), 0, total_rows, stream);

        // Return min and max device strings columns
        return {
          cudf::make_strings_column(
            total_rows,
            std::make_unique<column>(std::move(min_offsets), rmm::device_buffer{0, stream, mr}, 0),
            std::move(min_data),
            min_nulls,
            std::move(min_nullmask)),
          cudf::make_strings_column(
            total_rows,
            std::make_unique<column>(std::move(max_offsets), rmm::device_buffer{0, stream, mr}, 0),
            std::move(max_data),
            max_nulls,
            std::move(max_nullmask))};
      }
    }
  }
};

/**
 * @brief Custom CUDA kernel using Cooperative Groups to perform the paired logical OR reduction.
 * * NOTE: This operation is a map/stride-2-read, not a true block-to-global reduction.
 * CUB's BlockReduce is unsuitable here as it reduces a block to a single element.
 * Cooperative Groups is used here for robust global thread ID calculation.
 */
struct compute_next_level_functor {
  bool** const level_ptrs;
  cudf::size_type const current_level;
  cudf::size_type const current_level_size;
  cudf::size_type const next_level_size;

  __device__ void operator()(cudf::size_type next_level_index) const noexcept
  {
    auto const current_level_ptr = level_ptrs[current_level];
    auto next_level_ptr          = level_ptrs[current_level + 1];

    // Handle the odd-sized remaining element if current_level_size is odd
    if (current_level_size % 2 and next_level_index == (next_level_size - 1)) {
      // The last element is carried forward (ORed with false)
      next_level_ptr[next_level_index] = current_level_ptr[current_level_size - 1];
    } else {
      // Perform the logical OR reduction and write to the next level's location
      next_level_ptr[next_level_index] =
        current_level_ptr[(next_level_index * 2)] or current_level_ptr[(next_level_index * 2) + 1];
    }
  }
};

/**
 * @brief CUDA kernel to probe multiple ranges against the pre-calculated mask hierarchy.
 * One thread handles the binary decomposition and query for one range [M, N).
 * * @param d_level_ptrs Device array of pointers, where d_level_ptrs[k] points to the start of
 * Level k mask.
 * @param d_range_offsets Device array where range i is [d_range_offsets[i], d_range_offsets[i+1]).
 * @param num_ranges The number of ranges to process.
 * @param d_results Pointer to device memory to store the boolean result (true if a '1' is found in
 * the range).
 */
struct probe_masks_functor {
  bool** const level_ptrs;
  cudf::size_type const* const page_offsets;
  cudf::size_type const num_ranges;

  __device__ bool operator()(cudf::size_type range_idx) const noexcept
  {
    // Retrieve M and N for the current range [M, N)
    size_type M = page_offsets[range_idx];
    size_type N = page_offsets[range_idx + 1];

    // If the range is empty or invalid, terminate
    if (M >= N) { return false; }

    // Binary Decomposition Loop
    while (M < N) {
      // 1. Calculate the largest power of 2 that can align M up to the boundary.
      // This is determined by the Least Significant Bit (LSB) of M.
      // If M=0, LSB is usually defined as the full size, but here M is typically > 0
      // or we handle M=0 implicitly by the full range check.
      // The expression (M & -M) gives the value of the LSB, which is the block size (2^k).
      size_t m_lsb_block_size = (M == 0) ? N : (M & -M);
      size_t m_next_aligned   = M + m_lsb_block_size;

      // 2. Calculate the largest power of 2 block that can align N down to the boundary.
      // This is determined by the LSB of (N - M), but simpler to use N's alignment for the end.
      // The expression (N & -N) gives the block size corresponding to N's alignment.
      // We ensure N_lsb_block_size does not exceed the remaining range size (N-M).
      size_t n_lsb_block_size = N & -N;

      // --- Decision Logic: Which side to consume? ---

      // Block 1: M-aligned block (from M up to m_next_aligned)
      size_t block1_size = m_next_aligned - M;

      // Block 2: N-aligned block (from N - n_lsb_block_size up to N)
      size_t block2_size = n_lsb_block_size;

      // Block 3: The remaining central range block

      if (block1_size > 0 && M < m_next_aligned && m_next_aligned <= N) {
        // If the M-aligned block is fully contained in the range [M, N)

        // Check if block1_size is 2^k. k = log2(block1_size).
        // Since block1_size is based on LSB, it is always a power of 2.
        size_t k1 = __ffs(block1_size) - 1;

        // Calculate mask index: The starting point M is divided by the block size.
        size_t mask_idx = M / block1_size;

        // Look up the mask value
        if (level_ptrs[k1][mask_idx]) {
          return true;  // Found a set bit, terminate for this range
        }

        // Advance M
        M = m_next_aligned;
      } else if (block2_size > 0 && N - block2_size >= M) {
        // If the N-aligned block is fully contained and does not overlap M's new position

        // Check if block2_size is 2^k. k = log2(block2_size).
        size_t k2 = __ffs(block2_size) - 1;

        // Calculate mask index
        size_t mask_idx = (N - block2_size) / block2_size;

        // Look up the mask value
        if (level_ptrs[k2][mask_idx]) {
          return true;  // Found a set bit, terminate for this range
        }

        // Backtrack N
        N = N - block2_size;
      } else {
        // The remaining range is unaligned and small (or just 1 element).
        // This happens when M and N are close and unaligned (e.g., [11, 13]).

        // Prioritize M (1-row check) or N (1-row check) until they meet.

        // Check single row at M (Level 0)
        if (level_ptrs[0][M]) { return true; }
        M++;
      }
    }
    return false;
  }
};

}  // namespace

std::unique_ptr<cudf::column> aggregate_reader_metadata::build_row_mask_with_page_index_stats(
  cudf::host_span<std::vector<size_type> const> row_group_indices,
  cudf::host_span<cudf::data_type const> output_dtypes,
  cudf::host_span<cudf::size_type const> output_column_schemas,
  std::reference_wrapper<ast::expression const> filter,
  rmm::cuda_stream_view stream,
  rmm::device_async_resource_ref mr) const
{
  CUDF_FUNC_RANGE();

  // Return if empty row group indices
  if (row_group_indices.empty()) { return cudf::make_empty_column(cudf::type_id::BOOL8); }

  // Check if we have page index for all columns in all row groups
  auto const has_page_index = compute_has_page_index(per_file_metadata, row_group_indices);

  // Return if page index is not present
  CUDF_EXPECTS(has_page_index,
               "Page pruning requires the Parquet page index for all output columns",
               std::runtime_error);

  // Total number of rows
  auto const total_rows = std::accumulate(
    thrust::counting_iterator<size_t>(0),
    thrust::counting_iterator(row_group_indices.size()),
    size_t{0},
    [&](auto sum, auto const src_index) {
      auto const& rg_indices = row_group_indices[src_index];
      return std::accumulate(
        rg_indices.begin(), rg_indices.end(), sum, [&](auto subsum, auto const rg_index) {
          CUDF_EXPECTS(subsum + per_file_metadata[src_index].row_groups[rg_index].num_rows <=
                         std::numeric_limits<size_type>::max(),
                       "Total rows exceed the maximum value");
          return subsum + per_file_metadata[src_index].row_groups[rg_index].num_rows;
        });
    });

  auto const num_columns = output_dtypes.size();

  // Get a boolean mask indicating which columns will participate in stats based filtering
  auto const stats_columns_mask =
    parquet::detail::stats_columns_collector{filter.get(),
                                             static_cast<size_type>(output_dtypes.size())}
      .get_stats_columns_mask();

  // Return early if no columns will participate in stats based page filtering
  if (stats_columns_mask.empty()) {
    auto const scalar_true = cudf::numeric_scalar<bool>(true, true, stream);
    return cudf::make_column_from_scalar(scalar_true, total_rows, stream, mr);
  }

  // Convert page statistics to a table
  // where min(col[i]) = columns[i*2], max(col[i])=columns[i*2+1]
  // For each column, it contains total number of rows from all row groups.
  page_stats_caster const stats_col{
    static_cast<size_type>(total_rows), per_file_metadata, row_group_indices};

  std::vector<std::unique_ptr<column>> columns;
  std::for_each(
    thrust::counting_iterator<size_t>(0),
    thrust::counting_iterator(num_columns),
    [&](auto col_idx) {
      auto const schema_idx = output_column_schemas[col_idx];
      auto const& dtype     = output_dtypes[col_idx];
      // Only participating columns and comparable types except fixed point are supported
      if (not stats_columns_mask[col_idx] or
          (cudf::is_compound(dtype) && dtype.id() != cudf::type_id::STRING)) {
        // Placeholder for unsupported types and non-participating columns
        columns.push_back(cudf::make_numeric_column(
          data_type{cudf::type_id::BOOL8}, total_rows, rmm::device_buffer{}, 0, stream, mr));
        columns.push_back(cudf::make_numeric_column(
          data_type{cudf::type_id::BOOL8}, total_rows, rmm::device_buffer{}, 0, stream, mr));
        return;
      }
      auto [min_col, max_col] = cudf::type_dispatcher<dispatch_storage_type>(
        dtype, stats_col, schema_idx, dtype, stream, mr);
      columns.push_back(std::move(min_col));
      columns.push_back(std::move(max_col));
    });

  auto stats_table = cudf::table(std::move(columns));

  // Converts AST to StatsAST with reference to min, max columns in above `stats_table`.
  parquet::detail::stats_expression_converter const stats_expr{
    filter.get(), static_cast<size_type>(output_dtypes.size()), stream};

  // Filter the input table using AST expression and return the (BOOL8) predicate column.
  return cudf::detail::compute_column(stats_table, stats_expr.get_stats_expr().get(), stream, mr);
}

template <typename ColumnView>
cudf::detail::host_vector<bool> aggregate_reader_metadata::compute_data_page_mask(
  ColumnView const& row_mask,
  cudf::host_span<std::vector<size_type> const> row_group_indices,
  cudf::host_span<input_column_info const> input_columns,
  cudf::size_type row_mask_offset,
  rmm::cuda_stream_view stream) const
{
  CUDF_FUNC_RANGE();

  CUDF_EXPECTS(row_mask.type().id() == cudf::type_id::BOOL8,
               "Input row bitmask should be of type BOOL8");

  auto const total_rows = total_rows_in_row_groups(row_group_indices);

  // Return an empty vector if all rows are invalid or all rows are required
  if (row_mask.null_count(row_mask_offset, row_mask_offset + total_rows, stream) == total_rows or
      thrust::all_of(rmm::exec_policy(stream),
                     row_mask.template begin<bool>() + row_mask_offset,
                     row_mask.template begin<bool>() + row_mask_offset + total_rows,
                     cuda::std::identity{})) {
    return cudf::detail::make_empty_host_vector<bool>(0, stream);
  }

  CUDF_EXPECTS(row_mask_offset + total_rows <= row_mask.size(),
               "Mismatch in total rows in input row mask and row groups",
               std::invalid_argument);

  auto const num_columns = input_columns.size();

  // Collect column schema indices from the input columns.
  auto column_schema_indices = std::vector<size_type>(input_columns.size());
  std::transform(
    input_columns.begin(), input_columns.end(), column_schema_indices.begin(), [](auto const& col) {
      return col.schema_idx;
    });
  auto const has_page_index = compute_has_page_index(per_file_metadata, row_group_indices);

  // Return early if page index is not present
  if (not has_page_index) {
    CUDF_LOG_WARN("Encountered missing Parquet page index for one or more output columns");
    return cudf::detail::make_empty_host_vector<bool>(
      0, stream);  // An empty data page mask indicates all pages are required
  }

  // Compute page row offsets and column chunk page offsets for each column
  std::vector<size_type> page_row_offsets;
  std::vector<size_type> col_page_offsets;
  col_page_offsets.reserve(num_columns + 1);
  col_page_offsets.push_back(0);

  size_type max_page_size = 0;

  if (num_columns == 1) {
    auto const schema_idx   = column_schema_indices.front();
    size_type col_num_pages = 0;
    std::tie(page_row_offsets, col_num_pages, max_page_size) =
      compute_page_row_offsets(per_file_metadata, row_group_indices, schema_idx);
    // Add 1 to include the the 0th page's offset for each column
    col_page_offsets.emplace_back(col_num_pages + 1);
  } else {
    std::vector<std::future<std::tuple<std::vector<size_type>, size_type, size_type>>>
      page_row_offsets_tasks;
    page_row_offsets_tasks.reserve(num_columns);

    std::for_each(thrust::counting_iterator<size_t>(0),
                  thrust::counting_iterator(num_columns),
                  [&](auto const col_idx) {
                    page_row_offsets_tasks.emplace_back(
                      cudf::detail::host_worker_pool().submit_task([&, col_idx = col_idx] {
                        return compute_page_row_offsets(
                          per_file_metadata, row_group_indices, column_schema_indices[col_idx]);
                      }));
                  });

    // Collect results from all tasks
    std::for_each(page_row_offsets_tasks.begin(), page_row_offsets_tasks.end(), [&](auto& task) {
      auto [col_page_row_offsets, col_num_pages, col_max_page_size] = std::move(task).get();
      page_row_offsets.insert(page_row_offsets.end(),
                              std::make_move_iterator(col_page_row_offsets.begin()),
                              std::make_move_iterator(col_page_row_offsets.end()));
      max_page_size = std::max<size_type>(max_page_size, col_max_page_size);
      // Add 1 to include the the 0th page's offset for each column
      col_page_offsets.emplace_back(col_page_offsets.back() + col_num_pages + 1);
    });
  }

  auto const total_pages = page_row_offsets.size() - num_columns;

  // Make sure all row_mask elements contain valid values even if they are nulls
  if constexpr (cuda::std::is_same_v<ColumnView, cudf::mutable_column_view>) {
    if (row_mask.nullable()) {
      thrust::for_each(rmm::exec_policy_nosync(stream),
                       thrust::counting_iterator(row_mask_offset),
                       thrust::counting_iterator(row_mask_offset + total_rows),
                       [row_mask  = row_mask.template begin<bool>(),
                        null_mask = row_mask.null_mask()] __device__(auto const row_idx) {
                         if (not bit_is_set(null_mask, row_idx)) { row_mask[row_idx] = true; }
                       });
    }
  } else {
    CUDF_EXPECTS(not row_mask.nullable() or row_mask.null_count() == 0,
                 "Row mask must not contain nulls for payload columns");
  }

  auto const mr = cudf::get_current_device_resource_ref();
  auto const [level_offsets, total_levels_size] =
    compute_row_mask_levels(total_rows, max_page_size);
  auto const num_levels = static_cast<cudf::size_type>(level_offsets.size());

  auto levels_data = rmm::device_uvector<bool>(total_levels_size, stream, mr);

  auto host_level_ptrs = cudf::detail::make_host_vector<bool*>(num_levels, stream);
  host_level_ptrs[0]   = const_cast<bool*>(row_mask.template begin<bool>()) + row_mask_offset;
  std::for_each(
    thrust::counting_iterator(1), thrust::counting_iterator(num_levels), [&](auto const level_idx) {
      host_level_ptrs[level_idx] = levels_data.data() + level_offsets[level_idx - 1];
    });

  auto device_level_ptrs  = cudf::detail::make_device_uvector_async(host_level_ptrs, stream, mr);
  auto current_level_size = total_rows;
  std::for_each(
    thrust::counting_iterator(0), thrust::counting_iterator(num_levels - 1), [&](auto const level) {
      auto const next_level_size = cudf::util::div_rounding_up_unsafe(current_level_size, 2);
      thrust::for_each(rmm::exec_policy_nosync(stream),
                       thrust::counting_iterator(0),
                       thrust::counting_iterator(next_level_size),
                       compute_next_level_functor{
                         device_level_ptrs.data(), level, current_level_size, next_level_size});
      current_level_size = next_level_size;
    });

  auto const num_ranges = static_cast<cudf::size_type>(page_row_offsets.size() - 1);
  rmm::device_uvector<bool> device_data_page_mask(num_ranges, stream, mr);
  auto page_offsets = cudf::detail::make_device_uvector_async(page_row_offsets, stream, mr);
  thrust::transform(rmm::exec_policy_nosync(stream),
                    thrust::counting_iterator(0),
                    thrust::counting_iterator(num_ranges),
                    device_data_page_mask.begin(),
                    probe_masks_functor{device_level_ptrs.data(), page_offsets.data(), num_ranges});

  auto host_results      = cudf::detail::make_host_vector_async(device_data_page_mask, stream);
  auto data_page_mask    = cudf::detail::make_empty_host_vector<bool>(total_pages, stream);
  auto host_results_iter = host_results.begin();
  stream.synchronize();
  std::for_each(thrust::counting_iterator<size_t>(0),
                thrust::counting_iterator(num_columns),
                [&](auto col_idx) {
                  auto const col_num_pages =
                    col_page_offsets[col_idx + 1] - col_page_offsets[col_idx] - 1;
                  data_page_mask.insert(
                    data_page_mask.end(), host_results_iter, host_results_iter + col_num_pages);
                  host_results_iter += col_num_pages + 1;
                });
  return data_page_mask;
}

// Instantiate the templates with ColumnView as cudf::column_view and cudf::mutable_column_view
template cudf::detail::host_vector<bool> aggregate_reader_metadata::compute_data_page_mask<
  cudf::column_view>(cudf::column_view const& row_mask,
                     cudf::host_span<std::vector<size_type> const> row_group_indices,
                     cudf::host_span<input_column_info const> input_columns,
                     cudf::size_type row_mask_offset,
                     rmm::cuda_stream_view stream) const;

template cudf::detail::host_vector<bool> aggregate_reader_metadata::compute_data_page_mask<
  cudf::mutable_column_view>(cudf::mutable_column_view const& row_mask,
                             cudf::host_span<std::vector<size_type> const> row_group_indices,
                             cudf::host_span<input_column_info const> input_columns,
                             cudf::size_type row_mask_offset,
                             rmm::cuda_stream_view stream) const;

}  // namespace cudf::io::parquet::experimental::detail
