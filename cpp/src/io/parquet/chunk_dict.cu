#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2021-2022, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "parquet_gpu.cuh"

#include <cudf/detail/iterator.cuh>
#include <cudf/detail/utilities/cuda.cuh>
#include <cudf/table/row_operators.cuh>

#include <rmm/exec_policy.hpp>

#include <cuda/atomic>

namespace cudf {
namespace io {
namespace parquet {
namespace gpu {
namespace {
constexpr int DEFAULT_BLOCK_SIZE = 256;
}

template <int block_size>
__global__ void __launch_bounds__(block_size)
  initialize_chunk_hash_maps_kernel(device_span<EncColumnChunk> chunks)
{
  auto chunk = chunks[blockIdx.x];
  auto t     = threadIdx.x;
  // fut: Now that per-chunk dict is same size as ck.num_values, try to not use one block per chunk
  for (size_type i = 0; i < chunk.dict_map_size; i += block_size) {
    if (t + i < chunk.dict_map_size) {
      new (&chunk.dict_map_slots[t + i].first) map_type::atomic_key_type{KEY_SENTINEL};
      new (&chunk.dict_map_slots[t + i].second) map_type::atomic_mapped_type{VALUE_SENTINEL};
    }
  }
}

template <typename T>
struct equality_functor {
  column_device_view const& col;
  __device__ bool operator()(size_type lhs_idx, size_type rhs_idx)
  {
    // We don't call this for nulls so this is fine
    return equality_compare(col.element<T>(lhs_idx), col.element<T>(rhs_idx));
  }
};

template <typename T>
struct hash_functor {
  column_device_view const& col;
  __device__ auto operator()(size_type idx) const
  {
    return cudf::detail::MurmurHash3_32<T>{}(col.element<T>(idx));
  }
};

struct map_insert_fn {
  map_type::device_mutable_view& map;

  template <typename T>
  __device__ bool operator()(column_device_view const& col, size_type i)
  {
    if constexpr (column_device_view::has_element_accessor<T>()) {
      auto hash_fn     = hash_functor<T>{col};
      auto equality_fn = equality_functor<T>{col};
      return map.insert(std::pair(i, i), hash_fn, equality_fn);
    } else {
      CUDF_UNREACHABLE("Unsupported type to insert in map");
    }
  }
};

struct map_find_fn {
  map_type::device_view& map;

  template <typename T>
  __device__ map_type::device_view::iterator operator()(column_device_view const& col, size_type i)
  {
    if constexpr (column_device_view::has_element_accessor<T>()) {
      auto hash_fn     = hash_functor<T>{col};
      auto equality_fn = equality_functor<T>{col};
      return map.find(i, hash_fn, equality_fn);
    } else {
      CUDF_UNREACHABLE("Unsupported type to find in map");
    }
  }
};

template <int block_size>
__global__ void __launch_bounds__(block_size)
  populate_chunk_hash_maps_kernel(cudf::detail::device_2dspan<gpu::PageFragment const> frags)
{
  auto col_idx = blockIdx.y;
  auto block_x = blockIdx.x;
  auto t       = threadIdx.x;
  auto frag    = frags[col_idx][block_x];
  auto chunk   = frag.chunk;
  auto col     = chunk->col_desc;

  if (not chunk->use_dictionary) { return; }

  using block_reduce = hipcub::BlockReduce<size_type, block_size>;
  __shared__ typename block_reduce::TempStorage reduce_storage;

  size_type start_row = frag.start_row;
  size_type end_row   = frag.start_row + frag.num_rows;

  // Find the bounds of values in leaf column to be inserted into the map for current chunk
  size_type const s_start_value_idx = row_to_value_idx(start_row, *col);
  size_type const end_value_idx     = row_to_value_idx(end_row, *col);

  column_device_view const& data_col = *col->leaf_column;

  // Make a view of the hash map
  auto hash_map_mutable =
    map_type::device_mutable_view(chunk->dict_map_slots,
                                  chunk->dict_map_size,
                                  cuco::sentinel::empty_key{KEY_SENTINEL},
                                  cuco::sentinel::empty_value{VALUE_SENTINEL});

  __shared__ size_type total_num_dict_entries;
  size_type val_idx = s_start_value_idx + t;
  while (val_idx - block_size < end_value_idx) {
    auto const is_valid =
      val_idx < end_value_idx and val_idx < data_col.size() and data_col.is_valid(val_idx);

    // insert element at val_idx to hash map and count successful insertions
    size_type is_unique      = 0;
    size_type uniq_elem_size = 0;
    if (is_valid) {
      is_unique =
        type_dispatcher(data_col.type(), map_insert_fn{hash_map_mutable}, data_col, val_idx);
      uniq_elem_size = [&]() -> size_type {
        if (not is_unique) { return 0; }
        switch (col->physical_type) {
          case Type::INT32: return 4;
          case Type::INT64: return 8;
          case Type::INT96: return 12;
          case Type::FLOAT: return 4;
          case Type::DOUBLE: return 8;
          case Type::BYTE_ARRAY: {
            auto const col_type = data_col.type().id();
            if (col_type == type_id::STRING) {
              // Strings are stored as 4 byte length + string bytes
              return 4 + data_col.element<string_view>(val_idx).size_bytes();
            } else if (col_type == type_id::LIST) {
              // Binary is stored as 4 byte length + bytes
              return 4 + get_element<statistics::byte_array_view>(data_col, val_idx).size_bytes();
            }
            CUDF_UNREACHABLE(
              "Byte array only supports string and list<byte> column types for dictionary "
              "encoding!");
          }
          case Type::FIXED_LEN_BYTE_ARRAY:
            if (data_col.type().id() == type_id::DECIMAL128) { return sizeof(__int128_t); }
            CUDF_UNREACHABLE(
              "Fixed length byte array only supports decimal 128 column types for dictionary "
              "encoding!");
          default: CUDF_UNREACHABLE("Unsupported type for dictionary encoding");
        }
      }();
    }

    auto num_unique = block_reduce(reduce_storage).Sum(is_unique);
    __syncthreads();
    auto uniq_data_size = block_reduce(reduce_storage).Sum(uniq_elem_size);
    if (t == 0) {
      total_num_dict_entries = atomicAdd(&chunk->num_dict_entries, num_unique);
      total_num_dict_entries += num_unique;
      atomicAdd(&chunk->uniq_data_size, uniq_data_size);
    }
    __syncthreads();

    // Check if the num unique values in chunk has already exceeded max dict size and early exit
    if (total_num_dict_entries > MAX_DICT_SIZE) { return; }

    val_idx += block_size;
  }  // while
}

template <int block_size>
__global__ void __launch_bounds__(block_size)
  collect_map_entries_kernel(device_span<EncColumnChunk> chunks)
{
  auto& chunk = chunks[blockIdx.x];
  if (not chunk.use_dictionary) { return; }

  auto t   = threadIdx.x;
  auto map = map_type::device_view(chunk.dict_map_slots,
                                   chunk.dict_map_size,
                                   cuco::sentinel::empty_key{KEY_SENTINEL},
                                   cuco::sentinel::empty_value{VALUE_SENTINEL});

  __shared__ cuda::atomic<size_type, cuda::thread_scope_block> counter;
  using cuda::std::memory_order_relaxed;
  if (t == 0) { new (&counter) cuda::atomic<size_type, cuda::thread_scope_block>{0}; }
  __syncthreads();
  for (size_type i = 0; i < chunk.dict_map_size; i += block_size) {
    if (t + i < chunk.dict_map_size) {
      auto* slot = reinterpret_cast<map_type::value_type*>(map.begin_slot() + t + i);
      auto key   = slot->first;
      if (key != KEY_SENTINEL) {
        auto loc = counter.fetch_add(1, memory_order_relaxed);
        cudf_assert(loc < MAX_DICT_SIZE && "Number of filled slots exceeds max dict size");
        chunk.dict_data[loc] = key;
        // If sorting dict page ever becomes a hard requirement, enable the following statement and
        // add a dict sorting step before storing into the slot's second field.
        // chunk.dict_data_idx[loc] = t + i;
        slot->second = loc;
      }
    }
  }
}

template <int block_size>
__global__ void __launch_bounds__(block_size)
  get_dictionary_indices_kernel(cudf::detail::device_2dspan<gpu::PageFragment const> frags)
{
  auto col_idx = blockIdx.y;
  auto block_x = blockIdx.x;
  auto t       = threadIdx.x;
  auto frag    = frags[col_idx][block_x];
  auto chunk   = frag.chunk;
  auto col     = chunk->col_desc;

  if (not chunk->use_dictionary) { return; }

  size_type start_row = frag.start_row;
  size_type end_row   = frag.start_row + frag.num_rows;

  // Find the bounds of values in leaf column to be searched in the map for current chunk
  auto const s_start_value_idx  = row_to_value_idx(start_row, *col);
  auto const s_ck_start_val_idx = row_to_value_idx(chunk->start_row, *col);
  auto const end_value_idx      = row_to_value_idx(end_row, *col);

  column_device_view const& data_col = *col->leaf_column;

  auto map = map_type::device_view(chunk->dict_map_slots,
                                   chunk->dict_map_size,
                                   cuco::sentinel::empty_key{KEY_SENTINEL},
                                   cuco::sentinel::empty_value{VALUE_SENTINEL});

  auto val_idx = s_start_value_idx + t;
  while (val_idx < end_value_idx) {
    auto const is_valid = val_idx < data_col.size() and data_col.is_valid(val_idx);

    if (is_valid) {
      auto found_slot = type_dispatcher(data_col.type(), map_find_fn{map}, data_col, val_idx);
      cudf_assert(found_slot != map.end() &&
                  "Unable to find value in map in dictionary index construction");
      if (found_slot != map.end()) {
        // No need for atomic as this is not going to be modified by any other thread
        auto* val_ptr = reinterpret_cast<map_type::mapped_type*>(&found_slot->second);
        chunk->dict_index[val_idx - s_ck_start_val_idx] = *val_ptr;
      }
    }

    val_idx += block_size;
  }
}

void initialize_chunk_hash_maps(device_span<EncColumnChunk> chunks, rmm::cuda_stream_view stream)
{
  constexpr int block_size = 1024;
  initialize_chunk_hash_maps_kernel<block_size>
    <<<chunks.size(), block_size, 0, stream.value()>>>(chunks);
}

void populate_chunk_hash_maps(cudf::detail::device_2dspan<gpu::PageFragment const> frags,
                              rmm::cuda_stream_view stream)
{
  dim3 const dim_grid(frags.size().second, frags.size().first);
  populate_chunk_hash_maps_kernel<DEFAULT_BLOCK_SIZE>
    <<<dim_grid, DEFAULT_BLOCK_SIZE, 0, stream.value()>>>(frags);
}

void collect_map_entries(device_span<EncColumnChunk> chunks, rmm::cuda_stream_view stream)
{
  constexpr int block_size = 1024;
  collect_map_entries_kernel<block_size><<<chunks.size(), block_size, 0, stream.value()>>>(chunks);
}

void get_dictionary_indices(cudf::detail::device_2dspan<gpu::PageFragment const> frags,
                            rmm::cuda_stream_view stream)
{
  dim3 const dim_grid(frags.size().second, frags.size().first);
  get_dictionary_indices_kernel<DEFAULT_BLOCK_SIZE>
    <<<dim_grid, DEFAULT_BLOCK_SIZE, 0, stream.value()>>>(frags);
}
}  // namespace gpu
}  // namespace parquet
}  // namespace io
}  // namespace cudf
