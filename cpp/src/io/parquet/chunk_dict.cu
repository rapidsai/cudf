#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2021-2024, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "parquet_gpu.cuh"

#include <cudf/detail/iterator.cuh>
#include <cudf/detail/utilities/cuda.cuh>
#include <cudf/table/experimental/row_operators.cuh>

#include <rmm/exec_policy.hpp>

#include <cuco/static_map_ref.cuh>
#include <cuda/atomic>

namespace cudf::io::parquet::detail {

namespace {
constexpr int DEFAULT_BLOCK_SIZE = 256;
}

template <typename T>
struct equality_functor {
  column_device_view const& col;
  __device__ bool operator()(key_type lhs_idx, key_type rhs_idx) const
  {
    // We don't call this for nulls so this is fine.
    auto constexpr equal =
      cudf::experimental::row::equality::nan_equal_physical_equality_comparator{};
    return equal(col.element<T>(lhs_idx), col.element<T>(rhs_idx));
  }
};

template <typename T>
struct hash_functor {
  column_device_view const& col;
  uint32_t const seed = 0;
  __device__ auto operator()(key_type idx) const
  {
    return cudf::hashing::detail::MurmurHash3_x86_32<T>{seed}(col.element<T>(idx));
  }
};

template <int block_size>
struct map_insert_fn {
  storage_ref_type const& storage_ref;
  EncColumnChunk* const& chunk;

  template <typename T>
  __device__ void operator()(size_type const s_start_value_idx, size_type const end_value_idx)
  {
    if constexpr (column_device_view::has_element_accessor<T>()) {
      using block_reduce = hipcub::BlockReduce<size_type, block_size>;
      __shared__ typename block_reduce::TempStorage reduce_storage;

      auto const col                     = chunk->col_desc;
      column_device_view const& data_col = *col->leaf_column;
      __shared__ size_type total_num_dict_entries;

      using equality_fn_type = equality_functor<T>;
      using hash_fn_type     = hash_functor<T>;
      // Choosing `linear_probing` over `double_hashing` for slighhhtly better performance seen in
      // benchmarks.
      using probing_scheme_type = cuco::linear_probing<map_cg_size, hash_fn_type>;

      // Make a view of the hash map.
      auto hash_map_ref = cuco::static_map_ref{cuco::empty_key{KEY_SENTINEL},
                                               cuco::empty_value{VALUE_SENTINEL},
                                               equality_fn_type{data_col},
                                               probing_scheme_type{hash_fn_type{data_col}},
                                               cuco::thread_scope_block,
                                               storage_ref};

      // Create a map ref with `cuco::insert` operator
      auto map_insert_ref = hash_map_ref.rebind_operators(cuco::insert);
      auto const t        = threadIdx.x;

      // Create atomic refs to the current chunk's num_dict_entries and uniq_data_size
      cuda::atomic_ref<size_type, SCOPE> const chunk_num_dict_entries{chunk->num_dict_entries};
      cuda::atomic_ref<size_type, SCOPE> const chunk_uniq_data_size{chunk->uniq_data_size};

      // Note: Adjust the following loop to use `cg::tile<map_cg_size>` if needed in the future.
      for (thread_index_type val_idx = s_start_value_idx + t; val_idx - t < end_value_idx;
           val_idx += block_size) {
        size_type is_unique      = 0;
        size_type uniq_elem_size = 0;

        // Check if this index is valid.
        auto const is_valid =
          val_idx < end_value_idx and val_idx < data_col.size() and data_col.is_valid(val_idx);

        // Insert tile_val_idx to hash map and count successful insertions.
        if (is_valid) {
          // Insert the keys using a single thread for best performance for now.
          is_unique      = map_insert_ref.insert(cuco::pair{val_idx, val_idx});
          uniq_elem_size = [&]() -> size_type {
            if (not is_unique) { return 0; }
            switch (col->physical_type) {
              case Type::INT32: return 4;
              case Type::INT64: return 8;
              case Type::INT96: return 12;
              case Type::FLOAT: return 4;
              case Type::DOUBLE: return 8;
              case Type::BYTE_ARRAY: {
                auto const col_type = data_col.type().id();
                if (col_type == type_id::STRING) {
                  // Strings are stored as 4 byte length + string bytes
                  return 4 + data_col.element<string_view>(val_idx).size_bytes();
                } else if (col_type == type_id::LIST) {
                  // Binary is stored as 4 byte length + bytes
                  return 4 +
                         get_element<statistics::byte_array_view>(data_col, val_idx).size_bytes();
                }
                CUDF_UNREACHABLE(
                  "Byte array only supports string and list<byte> column types for dictionary "
                  "encoding!");
              }
              case Type::FIXED_LEN_BYTE_ARRAY:
                if (data_col.type().id() == type_id::DECIMAL128) { return sizeof(__int128_t); }
                CUDF_UNREACHABLE(
                  "Fixed length byte array only supports decimal 128 column types for dictionary "
                  "encoding!");
              default: CUDF_UNREACHABLE("Unsupported type for dictionary encoding");
            }
          }();
        }
        // Reduce num_unique and uniq_data_size from all tiles.
        auto num_unique = block_reduce(reduce_storage).Sum(is_unique);
        __syncthreads();
        auto uniq_data_size = block_reduce(reduce_storage).Sum(uniq_elem_size);
        // The first thread in the block atomically updates total num_unique and uniq_data_size
        if (t == 0) {
          total_num_dict_entries =
            chunk_num_dict_entries.fetch_add(num_unique, cuda::std::memory_order_relaxed);
          total_num_dict_entries += num_unique;
          chunk_uniq_data_size.fetch_add(uniq_data_size, cuda::std::memory_order_relaxed);
        }
        __syncthreads();

        // Check if the num unique values in chunk has already exceeded max dict size and early exit
        if (total_num_dict_entries > MAX_DICT_SIZE) { return; }
      }  // for loop
    } else {
      CUDF_UNREACHABLE("Unsupported type to insert in map");
    }
  }
};

template <int block_size>
struct map_find_fn {
  storage_ref_type const& storage_ref;
  EncColumnChunk* const& chunk;
  template <typename T>
  __device__ void operator()(size_type const s_start_value_idx,
                             size_type const end_value_idx,
                             size_type const s_ck_start_val_idx)
  {
    if constexpr (column_device_view::has_element_accessor<T>()) {
      auto const col                     = chunk->col_desc;
      column_device_view const& data_col = *col->leaf_column;

      using equality_fn_type = equality_functor<T>;
      using hash_fn_type     = hash_functor<T>;
      // Choosing `linear_probing` over `double_hashing` for slighhhtly better performance seen in
      // benchmarks.
      using probing_scheme_type = cuco::linear_probing<map_cg_size, hash_fn_type>;

      // Make a view of the hash map.
      auto hash_map_ref = cuco::static_map_ref{cuco::empty_key{KEY_SENTINEL},
                                               cuco::empty_value{VALUE_SENTINEL},
                                               equality_fn_type{data_col},
                                               probing_scheme_type{hash_fn_type{data_col}},
                                               cuco::thread_scope_block,
                                               storage_ref};

      // Create a map ref with `cuco::find` operator
      auto const map_find_ref = hash_map_ref.rebind_operators(cuco::find);
      auto const t            = threadIdx.x;

      // Note: Adjust the following loop to use `cg::tiles<map_cg_size>` if needed in the future.
      for (thread_index_type val_idx = s_start_value_idx + t; val_idx < end_value_idx;
           val_idx += block_size) {
        // Find the key using a single thread for best performance for now.
        if (data_col.is_valid(val_idx)) {
          auto const found_slot = map_find_ref.find(val_idx);
          // Fail if we didn't find the previously inserted key.
          cudf_assert(found_slot != map_find_ref.end() &&
                      "Unable to find value in map in dictionary index construction");
          // No need for atomic as this is not going to be modified by any other thread.
          chunk->dict_index[val_idx - s_ck_start_val_idx] = found_slot->second;
        }
      }
    } else {
      CUDF_UNREACHABLE("Unsupported type to find in map");
    }
  }
};

template <int block_size>
CUDF_KERNEL void __launch_bounds__(block_size)
  populate_chunk_hash_maps_kernel(device_span<window_type> const map_storage,
                                  cudf::detail::device_2dspan<PageFragment const> frags)
{
  auto const col_idx = blockIdx.y;
  auto const block_x = blockIdx.x;
  auto const frag    = frags[col_idx][block_x];
  auto chunk         = frag.chunk;
  auto col           = chunk->col_desc;

  if (not chunk->use_dictionary) { return; }

  size_type start_row = frag.start_row;
  size_type end_row   = frag.start_row + frag.num_rows;

  // Find the bounds of values in leaf column to be inserted into the map for current chunk.
  size_type const s_start_value_idx = row_to_value_idx(start_row, *col);
  size_type const end_value_idx     = row_to_value_idx(end_row, *col);

  column_device_view const& data_col = *col->leaf_column;
  storage_ref_type const storage_ref{chunk->dict_map_size,
                                     map_storage.data() + chunk->dict_map_offset};
  type_dispatcher(data_col.type(),
                  map_insert_fn<block_size>{storage_ref, chunk},
                  s_start_value_idx,
                  end_value_idx);
}

template <int block_size>
CUDF_KERNEL void __launch_bounds__(block_size)
  collect_map_entries_kernel(device_span<window_type> const map_storage,
                             device_span<EncColumnChunk> chunks)
{
  auto& chunk = chunks[blockIdx.x];
  if (not chunk.use_dictionary) { return; }

  auto t = threadIdx.x;
  __shared__ cuda::atomic<size_type, SCOPE> counter;
  using cuda::std::memory_order_relaxed;
  if (t == 0) { new (&counter) cuda::atomic<size_type, SCOPE>{0}; }
  __syncthreads();

  // Iterate over all windows in the map.
  for (; t < chunk.dict_map_size; t += block_size) {
    auto window = map_storage.data() + chunk.dict_map_offset + t;
    // Collect all slots from each window.
    for (auto& slot : *window) {
      auto const key = slot.first;
      if (key != KEY_SENTINEL) {
        auto const loc = counter.fetch_add(1, memory_order_relaxed);
        cudf_assert(loc < MAX_DICT_SIZE && "Number of filled slots exceeds max dict size");
        chunk.dict_data[loc] = key;
        // If sorting dict page ever becomes a hard requirement, enable the following statement
        // and add a dict sorting step before storing into the slot's second field.
        // chunk.dict_data_idx[loc] = idx;
        slot.second = loc;
      }
    }
  }
}

template <int block_size>
CUDF_KERNEL void __launch_bounds__(block_size)
  get_dictionary_indices_kernel(device_span<window_type> const map_storage,
                                cudf::detail::device_2dspan<PageFragment const> frags)
{
  auto const col_idx = blockIdx.y;
  auto const block_x = blockIdx.x;
  auto const frag    = frags[col_idx][block_x];
  auto chunk         = frag.chunk;

  if (not chunk->use_dictionary) { return; }

  size_type start_row = frag.start_row;
  size_type end_row   = frag.start_row + frag.num_rows;

  auto const col = chunk->col_desc;
  // Find the bounds of values in leaf column to be searched in the map for current chunk
  auto const s_start_value_idx  = row_to_value_idx(start_row, *col);
  auto const s_ck_start_val_idx = row_to_value_idx(chunk->start_row, *col);
  auto const end_value_idx      = row_to_value_idx(end_row, *col);

  column_device_view const& data_col = *col->leaf_column;
  storage_ref_type const storage_ref{chunk->dict_map_size,
                                     map_storage.data() + chunk->dict_map_offset};

  type_dispatcher(data_col.type(),
                  map_find_fn<block_size>{storage_ref, chunk},
                  s_start_value_idx,
                  end_value_idx,
                  s_ck_start_val_idx);
}

void populate_chunk_hash_maps(device_span<window_type> const map_storage,
                              cudf::detail::device_2dspan<PageFragment const> frags,
                              rmm::cuda_stream_view stream)
{
  dim3 const dim_grid(frags.size().second, frags.size().first);
  populate_chunk_hash_maps_kernel<DEFAULT_BLOCK_SIZE>
    <<<dim_grid, DEFAULT_BLOCK_SIZE, 0, stream.value()>>>(map_storage, frags);
}

void collect_map_entries(device_span<window_type> const map_storage,
                         device_span<EncColumnChunk> chunks,
                         rmm::cuda_stream_view stream)
{
  constexpr int block_size = 1024;
  collect_map_entries_kernel<block_size>
    <<<chunks.size(), block_size, 0, stream.value()>>>(map_storage, chunks);
}

void get_dictionary_indices(device_span<window_type> const map_storage,
                            cudf::detail::device_2dspan<PageFragment const> frags,
                            rmm::cuda_stream_view stream)
{
  dim3 const dim_grid(frags.size().second, frags.size().first);
  get_dictionary_indices_kernel<DEFAULT_BLOCK_SIZE>
    <<<dim_grid, DEFAULT_BLOCK_SIZE, 0, stream.value()>>>(map_storage, frags);
}
}  // namespace cudf::io::parquet::detail
