#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2023-2024, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "delta_binary.cuh"
#include "io/utilities/column_buffer.hpp"
#include "page_decode.cuh"

#include <cudf/hashing/detail/default_hash.cuh>

#include <rmm/exec_policy.hpp>

#include <thrust/reduce.h>

namespace cudf::io::parquet::detail {

namespace {

// # of threads we're decoding with
constexpr int preprocess_block_size = 512;

// the required number of runs in shared memory we will need to provide the
// rle_stream object
constexpr int rle_run_buffer_size = rle_stream_required_run_buffer_size<preprocess_block_size>();

// the size of the rolling batch buffer
constexpr int rolling_buf_size = LEVEL_DECODE_BUF_SIZE;

using unused_state_buf = page_state_buffers_s<0, 0, 0>;

/**
 * @brief Calculate string bytes for DELTA_LENGTH_BYTE_ARRAY encoded pages
 *
 * Result is valid only on thread 0.
 *
 * @param s The local page info
 * @param t Thread index
 */
__device__ size_type gpuDeltaLengthPageStringSize(page_state_s* s, int t)
{
  if (t == 0) {
    // find the beginning of char data
    delta_binary_decoder string_lengths;
    auto const* string_start = string_lengths.find_end_of_block(s->data_start, s->data_end);
    // distance is size of string data
    return static_cast<size_type>(std::distance(string_start, s->data_end));
  }
  return 0;
}

/**
 * @brief Calculate string bytes for DELTA_BYTE_ARRAY encoded pages
 *
 * This expects all threads in the thread block (preprocess_block_size).
 *
 * @param s The local page info
 * @param t Thread index
 */
__device__ size_type gpuDeltaPageStringSize(page_state_s* s, int t)
{
  using cudf::detail::warp_size;
  using WarpReduce = hipcub::WarpReduce<uleb128_t>;
  __shared__ typename WarpReduce::TempStorage temp_storage[2];

  __shared__ __align__(16) delta_binary_decoder prefixes;
  __shared__ __align__(16) delta_binary_decoder suffixes;

  int const lane_id = t % warp_size;
  int const warp_id = t / warp_size;

  if (t == 0) {
    auto const* suffix_start = prefixes.find_end_of_block(s->data_start, s->data_end);
    suffixes.init_binary_block(suffix_start, s->data_end);
  }
  __syncthreads();

  // two warps will traverse the prefixes and suffixes and sum them up
  auto const db = t < warp_size ? &prefixes : t < 2 * warp_size ? &suffixes : nullptr;

  size_t total_bytes = 0;
  if (db != nullptr) {
    // initialize with first value (which is stored in last_value)
    if (lane_id == 0) { total_bytes = db->last_value; }

    uleb128_t lane_sum = 0;
    while (db->current_value_idx < db->num_encoded_values(true)) {
      // calculate values for current mini-block
      db->calc_mini_block_values(lane_id);

      // get per lane sum for mini-block
      for (uint32_t i = 0; i < db->values_per_mb; i += warp_size) {
        uint32_t const idx = db->current_value_idx + i + lane_id;
        if (idx < db->value_count) {
          lane_sum += db->value[rolling_index<delta_rolling_buf_size>(idx)];
        }
      }

      if (lane_id == 0) { db->setup_next_mini_block(true); }
      __syncwarp();
    }

    // get sum for warp.
    // note: warp_sum will only be valid on lane 0.
    auto const warp_sum = WarpReduce(temp_storage[warp_id]).Sum(lane_sum);

    if (lane_id == 0) { total_bytes += warp_sum; }
  }
  __syncthreads();

  // now sum up total_bytes from the two warps. result is only valid on thread 0.
  auto const final_bytes =
    cudf::detail::single_lane_block_sum_reduce<preprocess_block_size, 0>(total_bytes);

  return static_cast<size_type>(final_bytes);
}

/**
 * @brief Calculate the number of string bytes in the page.
 *
 * This function expects the dictionary position to be at 0 and will traverse
 * the entire thing (for plain and dictionary encoding).
 *
 * This expects all threads in the thread block (preprocess_block_size). Result is only
 * valid on thread 0.
 *
 * @param s The local page info
 * @param t Thread index
 */
__device__ size_type gpuDecodeTotalPageStringSize(page_state_s* s, int t)
{
  using cudf::detail::warp_size;
  size_type target_pos = s->num_input_values;
  size_type str_len    = 0;
  switch (s->page.encoding) {
    case Encoding::PLAIN_DICTIONARY:
    case Encoding::RLE_DICTIONARY:
      if (t < warp_size && s->dict_base) {
        auto const [new_target_pos, len] =
          gpuDecodeDictionaryIndices<true, unused_state_buf>(s, nullptr, target_pos, t);
        target_pos = new_target_pos;
        str_len    = len;
      }
      break;

    case Encoding::PLAIN:
      // For V2 headers, we know how many values are present, so can skip an expensive scan.
      if ((s->page.flags & PAGEINFO_FLAGS_V2) != 0) {
        auto const num_values = s->page.num_input_values - s->page.num_nulls;
        str_len               = s->dict_size - sizeof(int) * num_values;
      }
      // For V1, the choice is an overestimate (s->dict_size), or an exact number that's
      // expensive to compute. For now we're going with the latter.
      else {
        str_len = gpuInitStringDescriptors<true, unused_state_buf>(s, nullptr, target_pos, t);
      }
      break;

    case Encoding::DELTA_LENGTH_BYTE_ARRAY: str_len = gpuDeltaLengthPageStringSize(s, t); break;

    case Encoding::DELTA_BYTE_ARRAY: str_len = gpuDeltaPageStringSize(s, t); break;

    default:
      // not a valid string encoding, so just return 0
      break;
  }
  if (!t) { s->dict_pos = target_pos; }
  return str_len;
}

/**
 * @brief Update output column sizes for every nesting level based on a batch
 * of incoming decoded definition and repetition level values.
 *
 * If bounds_set is true, computes skipped_values and skipped_leaf_values for the
 * page to indicate where we need to skip to based on min/max row.
 *
 * Operates at the block level.
 *
 * @param s The local page info
 * @param target_value_count The target value count to process up to
 * @param rep Repetition level buffer
 * @param def Definition level buffer
 * @param t Thread index
 * @param bounds_set A boolean indicating whether or not min/max row bounds have been set
 */
template <typename level_t>
static __device__ void gpuUpdatePageSizes(page_state_s* s,
                                          int target_value_count,
                                          level_t const* const rep,
                                          level_t const* const def,
                                          int t,
                                          bool bounds_set)
{
  // max nesting depth of the column
  int const max_depth = s->col.max_nesting_depth;

  constexpr int num_warps      = preprocess_block_size / 32;
  constexpr int max_batch_size = num_warps * 32;

  using block_reduce = hipcub::BlockReduce<int, preprocess_block_size>;
  using block_scan   = hipcub::BlockScan<int, preprocess_block_size>;
  __shared__ union {
    typename block_reduce::TempStorage reduce_storage;
    typename block_scan::TempStorage scan_storage;
  } temp_storage;

  // how many input level values we've processed in the page so far
  int value_count = s->input_value_count;
  // how many rows we've processed in the page so far
  int row_count = s->input_row_count;
  // how many leaf values we've processed in the page so far
  int leaf_count = s->input_leaf_count;
  // whether or not we need to continue checking for the first row
  bool skipped_values_set = s->page.skipped_values >= 0;

  while (value_count < target_value_count) {
    int const batch_size = min(max_batch_size, target_value_count - value_count);

    // start/end depth
    int start_depth, end_depth, d;
    get_nesting_bounds<rolling_buf_size, level_t>(
      start_depth, end_depth, d, s, rep, def, value_count, value_count + batch_size, t);

    // is this thread within row bounds? in the non skip_rows/num_rows case this will always
    // be true.
    int in_row_bounds = 1;

    // if we are in the skip_rows/num_rows case, we need to check against these limits
    if (bounds_set) {
      // get absolute thread row index
      int const is_new_row = start_depth == 0;
      int thread_row_count, block_row_count;
      block_scan(temp_storage.scan_storage)
        .InclusiveSum(is_new_row, thread_row_count, block_row_count);
      __syncthreads();

      // get absolute thread leaf index
      int const is_new_leaf = (d >= s->nesting_info[max_depth - 1].max_def_level);
      int thread_leaf_count, block_leaf_count;
      block_scan(temp_storage.scan_storage)
        .InclusiveSum(is_new_leaf, thread_leaf_count, block_leaf_count);
      __syncthreads();

      // if this thread is in row bounds
      int const row_index = (thread_row_count + row_count) - 1;
      in_row_bounds =
        (row_index >= s->row_index_lower_bound) && (row_index < (s->first_row + s->num_rows));

      // if we have not set skipped values yet, see if we found the first in-bounds row
      if (!skipped_values_set) {
        int local_count, global_count;
        block_scan(temp_storage.scan_storage)
          .InclusiveSum(in_row_bounds, local_count, global_count);
        __syncthreads();

        // we found it
        if (global_count > 0) {
          // this is the thread that represents the first row.
          if (local_count == 1 && in_row_bounds) {
            s->page.skipped_values = value_count + t;
            s->page.skipped_leaf_values =
              leaf_count + (is_new_leaf ? thread_leaf_count - 1 : thread_leaf_count);
          }
          skipped_values_set = true;
        }
      }

      row_count += block_row_count;
      leaf_count += block_leaf_count;
    }

    // increment value counts across all nesting depths
    for (int s_idx = 0; s_idx < max_depth; s_idx++) {
      int const in_nesting_bounds = (s_idx >= start_depth && s_idx <= end_depth && in_row_bounds);
      int const count = block_reduce(temp_storage.reduce_storage).Sum(in_nesting_bounds);
      __syncthreads();
      if (!t) {
        PageNestingInfo* pni = &s->page.nesting[s_idx];
        pni->batch_size += count;
      }
    }

    value_count += batch_size;
  }

  // update final outputs
  if (!t) {
    s->input_value_count = value_count;

    // only used in the skip_rows/num_rows case
    s->input_leaf_count = leaf_count;
    s->input_row_count  = row_count;
  }
}

/**
 * @brief Kernel for computing per-page column size information for all nesting levels.
 *
 * This function will write out the size field for each level of nesting.
 *
 * @param pages List of pages
 * @param chunks List of column chunks
 * @param min_row Row index to start reading at
 * @param num_rows Maximum number of rows to read. Pass as INT_MAX to guarantee reading all rows
 * @param is_base_pass Whether or not this is the base pass.  We first have to compute
 * the full size information of every page before we come through in a second (trim) pass
 * to determine what subset of rows in this page we should be reading
 * @param compute_string_sizes Whether or not we should be computing string sizes
 * (PageInfo::str_bytes) as part of the pass
 */
template <typename level_t>
CUDF_KERNEL void __launch_bounds__(preprocess_block_size)
  gpuComputePageSizes(PageInfo* pages,
                      device_span<ColumnChunkDesc const> chunks,
                      size_t min_row,
                      size_t num_rows,
                      bool is_base_pass,
                      bool compute_string_sizes)
{
  __shared__ __align__(16) page_state_s state_g;

  page_state_s* const s = &state_g;
  int page_idx          = blockIdx.x;
  int t                 = threadIdx.x;
  PageInfo* pp          = &pages[page_idx];

  // whether or not we have repetition levels (lists)
  bool has_repetition = chunks[pp->chunk_idx].max_level[level_type::REPETITION] > 0;

  // the level stream decoders
  __shared__ rle_run<level_t> def_runs[rle_run_buffer_size];
  __shared__ rle_run<level_t> rep_runs[rle_run_buffer_size];
  rle_stream<level_t, preprocess_block_size, rolling_buf_size>
    decoders[level_type::NUM_LEVEL_TYPES] = {{def_runs}, {rep_runs}};

  // setup page info
  if (!setupLocalPageInfo(
        s, pp, chunks, min_row, num_rows, all_types_filter{}, page_processing_stage::PREPROCESS)) {
    return;
  }

  // initialize the stream decoders (requires values computed in setupLocalPageInfo)
  // the size of the rolling batch buffer
  level_t* const rep = reinterpret_cast<level_t*>(pp->lvl_decode_buf[level_type::REPETITION]);
  level_t* const def = reinterpret_cast<level_t*>(pp->lvl_decode_buf[level_type::DEFINITION]);
  decoders[level_type::DEFINITION].init(s->col.level_bits[level_type::DEFINITION],
                                        s->abs_lvl_start[level_type::DEFINITION],
                                        s->abs_lvl_end[level_type::DEFINITION],
                                        def,
                                        s->page.num_input_values);
  if (has_repetition) {
    decoders[level_type::REPETITION].init(s->col.level_bits[level_type::REPETITION],
                                          s->abs_lvl_start[level_type::REPETITION],
                                          s->abs_lvl_end[level_type::REPETITION],
                                          rep,
                                          s->page.num_input_values);
  }
  __syncthreads();

  if (!t) {
    s->page.skipped_values      = -1;
    s->page.skipped_leaf_values = 0;
    // str_bytes_from_index will be 0 if no page stats are present
    s->page.str_bytes    = s->page.str_bytes_from_index;
    s->input_row_count   = 0;
    s->input_value_count = 0;

    // in the base pass, we're computing the number of rows, make sure we visit absolutely
    // everything
    if (is_base_pass) {
      s->first_row             = 0;
      s->num_rows              = INT_MAX;
      s->row_index_lower_bound = -1;
    }
  }

  // we only need to preprocess hierarchies with repetition in them (ie, hierarchies
  // containing lists anywhere within).
  compute_string_sizes =
    compute_string_sizes && s->col.physical_type == BYTE_ARRAY && !s->col.is_strings_to_cat;

  // early out optimizations:

  // - if this is a flat hierarchy (no lists) and is not a string column. in this case we don't need
  // to do the expensive work of traversing the level data to determine sizes.  we can just compute
  // it directly.
  if (!has_repetition && !compute_string_sizes) {
    int depth = 0;
    while (depth < s->page.num_output_nesting_levels) {
      auto const thread_depth = depth + t;
      if (thread_depth < s->page.num_output_nesting_levels) {
        if (is_base_pass) { pp->nesting[thread_depth].size = pp->num_input_values; }
        pp->nesting[thread_depth].batch_size = pp->num_input_values;
      }
      depth += blockDim.x;
    }
    return;
  }

  // in the trim pass, for anything with lists, we only need to fully process bounding pages (those
  // at the beginning or the end of the row bounds)
  if (!is_base_pass && !is_bounds_page(s, min_row, num_rows, has_repetition)) {
    int depth = 0;
    while (depth < s->page.num_output_nesting_levels) {
      auto const thread_depth = depth + t;
      if (thread_depth < s->page.num_output_nesting_levels) {
        // if we are not a bounding page (as checked above) then we are either
        // returning all rows/values from this page, or 0 of them
        pp->nesting[thread_depth].batch_size =
          (s->num_rows == 0 && !is_page_contained(s, min_row, num_rows))
            ? 0
            : pp->nesting[thread_depth].size;
      }
      depth += blockDim.x;
    }
    return;
  }

  // zero sizes
  int depth = 0;
  while (depth < s->page.num_output_nesting_levels) {
    auto const thread_depth = depth + t;
    if (thread_depth < s->page.num_output_nesting_levels) {
      s->page.nesting[thread_depth].batch_size = 0;
    }
    depth += blockDim.x;
  }
  __syncthreads();

  // the core loop. decode batches of level stream data using rle_stream objects
  // and pass the results to gpuUpdatePageSizes
  int processed = 0;
  while (processed < s->page.num_input_values) {
    // TODO:  it would not take much more work to make it so that we could run both of these
    // decodes concurrently. there are a couple of shared variables internally that would have to
    // get dealt with but that's about it.
    if (has_repetition) {
      decoders[level_type::REPETITION].decode_next(t);
      __syncthreads();
    }
    // the # of rep/def levels will always be the same size
    processed += decoders[level_type::DEFINITION].decode_next(t);
    __syncthreads();

    // update page sizes
    gpuUpdatePageSizes<level_t>(s, processed, rep, def, t, !is_base_pass);
    __syncthreads();
  }

  // retrieve total string size.
  if (compute_string_sizes && !pp->has_page_index) {
    auto const str_bytes = gpuDecodeTotalPageStringSize(s, t);
    if (t == 0) { s->page.str_bytes = str_bytes; }
  }

  // update output results:
  // - real number of rows for the whole page
  // - nesting sizes for the whole page
  // - skipped value information for trimmed pages
  // - string bytes
  if (is_base_pass) {
    // nesting level 0 is the root column, so the size is also the # of rows
    if (!t) { pp->num_rows = s->page.nesting[0].batch_size; }

    // store off this batch size as the "full" size
    int depth = 0;
    while (depth < s->page.num_output_nesting_levels) {
      auto const thread_depth = depth + t;
      if (thread_depth < s->page.num_output_nesting_levels) {
        pp->nesting[thread_depth].size = pp->nesting[thread_depth].batch_size;
      }
      depth += blockDim.x;
    }
  }

  if (!t) {
    pp->skipped_values      = s->page.skipped_values;
    pp->skipped_leaf_values = s->page.skipped_leaf_values;
    pp->str_bytes           = s->page.str_bytes;
  }
}

}  // anonymous namespace

/**
 * @copydoc cudf::io::parquet::gpu::ComputePageSizes
 */
void ComputePageSizes(cudf::detail::hostdevice_span<PageInfo> pages,
                      cudf::detail::hostdevice_span<ColumnChunkDesc const> chunks,
                      size_t min_row,
                      size_t num_rows,
                      bool compute_num_rows,
                      bool compute_string_sizes,
                      int level_type_size,
                      rmm::cuda_stream_view stream)
{
  dim3 dim_block(preprocess_block_size, 1);
  dim3 dim_grid(pages.size(), 1);  // 1 threadblock per page

  // computes:
  // PageNestingInfo::size for each level of nesting, for each page.
  // This computes the size for the entire page, not taking row bounds into account.
  // If uses_custom_row_bounds is set to true, we have to do a second pass later that "trims"
  // the starting and ending read values to account for these bounds.
  if (level_type_size == 1) {
    gpuComputePageSizes<uint8_t><<<dim_grid, dim_block, 0, stream.value()>>>(
      pages.device_ptr(), chunks, min_row, num_rows, compute_num_rows, compute_string_sizes);
  } else {
    gpuComputePageSizes<uint16_t><<<dim_grid, dim_block, 0, stream.value()>>>(
      pages.device_ptr(), chunks, min_row, num_rows, compute_num_rows, compute_string_sizes);
  }
}

}  // namespace cudf::io::parquet::detail
