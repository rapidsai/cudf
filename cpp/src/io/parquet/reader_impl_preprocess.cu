#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2022-2025, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "error.hpp"
#include "io/comp/common.hpp"
#include "reader_impl.hpp"
#include "reader_impl_preprocess_utils.cuh"

#include <cudf/detail/iterator.cuh>
#include <cudf/detail/nvtx/ranges.hpp>
#include <cudf/detail/utilities/batched_memset.hpp>
#include <cudf/detail/utilities/integer_utils.hpp>
#include <cudf/detail/utilities/vector_factories.hpp>
#include <cudf/utilities/memory_resource.hpp>

#include <rmm/exec_policy.hpp>

#include <thrust/fill.h>
#include <thrust/functional.h>
#include <thrust/iterator/transform_iterator.h>
#include <thrust/reduce.h>
#include <thrust/scan.h>
#include <thrust/transform.h>

#include <limits>
#include <numeric>

namespace cudf::io::parquet::detail {

void reader::impl::build_string_dict_indices()
{
  CUDF_FUNC_RANGE();

  auto& pass = *_pass_itm_data;

  // compute number of indices per chunk and a summed total
  rmm::device_uvector<size_t> str_dict_index_count(pass.chunks.size() + 1, _stream);
  thrust::fill(
    rmm::exec_policy_nosync(_stream), str_dict_index_count.begin(), str_dict_index_count.end(), 0);
  thrust::for_each(rmm::exec_policy_nosync(_stream),
                   pass.pages.d_begin(),
                   pass.pages.d_end(),
                   set_str_dict_index_count{str_dict_index_count, pass.chunks});

  size_t const total_str_dict_indexes = thrust::reduce(
    rmm::exec_policy(_stream), str_dict_index_count.begin(), str_dict_index_count.end());
  if (total_str_dict_indexes == 0) { return; }

  // convert to offsets
  rmm::device_uvector<size_t>& str_dict_index_offsets = str_dict_index_count;
  thrust::exclusive_scan(rmm::exec_policy_nosync(_stream),
                         str_dict_index_offsets.begin(),
                         str_dict_index_offsets.end(),
                         str_dict_index_offsets.begin(),
                         0);

  // allocate and distribute pointers
  pass.str_dict_index = cudf::detail::make_zeroed_device_uvector_async<string_index_pair>(
    total_str_dict_indexes, _stream, cudf::get_current_device_resource_ref());

  auto iter = thrust::make_counting_iterator(0);
  thrust::for_each(
    rmm::exec_policy_nosync(_stream),
    iter,
    iter + pass.chunks.size(),
    set_str_dict_index_ptr{pass.str_dict_index.data(), str_dict_index_offsets, pass.chunks});

  // compute the indices
  build_string_dictionary_index(pass.chunks.device_ptr(), pass.chunks.size(), _stream);
  pass.chunks.device_to_host(_stream);
}

void reader::impl::allocate_nesting_info()
{
  auto& pass    = *_pass_itm_data;
  auto& subpass = *pass.subpass;

  auto const num_columns         = _input_columns.size();
  auto& pages                    = subpass.pages;
  auto& page_nesting_info        = subpass.page_nesting_info;
  auto& page_nesting_decode_info = subpass.page_nesting_decode_info;

  // generate the number of nesting info structs needed per-page, by column
  std::vector<int> per_page_nesting_info_size(num_columns);
  auto iter = thrust::make_counting_iterator(size_type{0});
  std::transform(iter, iter + num_columns, per_page_nesting_info_size.begin(), [&](size_type i) {
    // Schema index of the current input column
    auto const schema_idx = _input_columns[i].schema_idx;
    // Get the max_definition_level of this column across all sources.
    auto max_definition_level = _metadata->get_schema(schema_idx).max_definition_level + 1;
    std::for_each(thrust::make_counting_iterator(static_cast<size_t>(1)),
                  thrust::make_counting_iterator(_sources.size()),
                  [&](auto const src_file_idx) {
                    auto const& schema = _metadata->get_schema(
                      _metadata->map_schema_index(schema_idx, src_file_idx), src_file_idx);
                    max_definition_level =
                      std::max(max_definition_level, schema.max_definition_level + 1);
                  });

    return std::max(max_definition_level, _metadata->get_output_nesting_depth(schema_idx));
  });

  // compute total # of page_nesting infos needed and allocate space. doing this in one
  // buffer to keep it to a single gpu allocation
  auto counting_iter = thrust::make_counting_iterator(size_t{0});
  size_t const total_page_nesting_infos =
    std::accumulate(counting_iter, counting_iter + num_columns, 0, [&](int total, size_t index) {
      return total + (per_page_nesting_info_size[index] * subpass.column_page_count[index]);
    });

  page_nesting_info =
    cudf::detail::hostdevice_vector<PageNestingInfo>{total_page_nesting_infos, _stream};
  page_nesting_decode_info =
    cudf::detail::hostdevice_vector<PageNestingDecodeInfo>{total_page_nesting_infos, _stream};

  // update pointers in the PageInfos
  int target_page_index = 0;
  int src_info_index    = 0;
  for (size_t idx = 0; idx < _input_columns.size(); idx++) {
    auto const src_col_schema = _input_columns[idx].schema_idx;

    for (size_t p_idx = 0; p_idx < subpass.column_page_count[idx]; p_idx++) {
      pages[target_page_index + p_idx].nesting = page_nesting_info.device_ptr() + src_info_index;
      pages[target_page_index + p_idx].nesting_decode =
        page_nesting_decode_info.device_ptr() + src_info_index;

      pages[target_page_index + p_idx].nesting_info_size = per_page_nesting_info_size[idx];
      // Set the number of output nesting levels from the zeroth source as nesting must be
      // identical across sources.
      pages[target_page_index + p_idx].num_output_nesting_levels =
        _metadata->get_output_nesting_depth(src_col_schema);

      src_info_index += per_page_nesting_info_size[idx];
    }
    target_page_index += subpass.column_page_count[idx];
  }

  // Reset the target_page_index
  target_page_index = 0;

  // fill in
  int nesting_info_index = 0;
  for (size_t idx = 0; idx < _input_columns.size(); idx++) {
    auto const src_col_schema = _input_columns[idx].schema_idx;

    // real depth of the output cudf column hierarchy (1 == no nesting, 2 == 1 level, etc)
    // nesting depth must be same across sources so getting it from the zeroth source is ok
    int const max_output_depth = _metadata->get_output_nesting_depth(src_col_schema);

    // Map to store depths if this column has lists
    std::map<std::pair<int, int>, std::pair<std::vector<int>, std::vector<int>>> depth_remapping;
    // if this column has lists, generate depth remapping
    std::for_each(
      thrust::make_counting_iterator(static_cast<size_t>(0)),
      thrust::make_counting_iterator(_sources.size()),
      [&](auto const src_file_idx) {
        auto const mapped_schema_idx = _metadata->map_schema_index(src_col_schema, src_file_idx);
        if (_metadata->get_schema(mapped_schema_idx, src_file_idx).max_repetition_level > 0) {
          generate_depth_remappings(
            depth_remapping, src_col_schema, mapped_schema_idx, src_file_idx, *_metadata);
        }
      });

    // fill in host-side nesting info
    int schema_idx = src_col_schema;
    // This is okay as we only use this to check stubness of cur_schema and
    // to get its parent's indices, both of which are one to one mapped.
    auto cur_schema = _metadata->get_schema(schema_idx);
    int cur_depth   = max_output_depth - 1;
    while (schema_idx > 0) {
      // stub columns (basically the inner field of a list schema element) are not real columns.
      // we can ignore them for the purposes of output nesting info
      if (!cur_schema.is_stub()) {
        // initialize each page within the chunk
        for (size_t p_idx = 0; p_idx < subpass.column_page_count[idx]; p_idx++) {
          // Source file index for the current page.
          auto const src_file_idx =
            pass.chunks[pages[target_page_index + p_idx].chunk_idx].src_file_idx;
          PageNestingInfo* pni =
            &page_nesting_info[nesting_info_index + (p_idx * per_page_nesting_info_size[idx])];

          PageNestingDecodeInfo* nesting_info =
            &page_nesting_decode_info[nesting_info_index +
                                      (p_idx * per_page_nesting_info_size[idx])];

          auto const mapped_src_col_schema =
            _metadata->map_schema_index(src_col_schema, src_file_idx);
          // if we have lists, set our start and end depth remappings
          if (_metadata->get_schema(mapped_src_col_schema, src_file_idx).max_repetition_level > 0) {
            auto remap = depth_remapping.find({src_col_schema, src_file_idx});
            CUDF_EXPECTS(remap != depth_remapping.end(),
                         "Could not find depth remapping for schema");
            std::vector<int> const& rep_depth_remap = (remap->second.first);
            std::vector<int> const& def_depth_remap = (remap->second.second);

            for (size_t m = 0; m < rep_depth_remap.size(); m++) {
              nesting_info[m].start_depth = rep_depth_remap[m];
            }
            for (size_t m = 0; m < def_depth_remap.size(); m++) {
              nesting_info[m].end_depth = def_depth_remap[m];
            }
          }

          // Get the schema from the current input source.
          auto& actual_cur_schema = _metadata->get_schema(
            _metadata->map_schema_index(schema_idx, src_file_idx), src_file_idx);

          // values indexed by output column index
          nesting_info[cur_depth].max_def_level = actual_cur_schema.max_definition_level;
          pni[cur_depth].size                   = 0;
          pni[cur_depth].type =
            to_type_id(actual_cur_schema, _strings_to_categorical, _options.timestamp_type.id());
          pni[cur_depth].nullable = cur_schema.repetition_type == FieldRepetitionType::OPTIONAL;
        }

        // move up the hierarchy
        cur_depth--;
      }

      // next schema
      schema_idx = cur_schema.parent_idx;
      cur_schema = _metadata->get_schema(schema_idx);
    }

    // Offset the page and nesting info indices
    target_page_index += subpass.column_page_count[idx];
    nesting_info_index += (per_page_nesting_info_size[idx] * subpass.column_page_count[idx]);
  }

  // copy nesting info to the device
  page_nesting_info.host_to_device_async(_stream);
  page_nesting_decode_info.host_to_device_async(_stream);
}

void reader::impl::allocate_level_decode_space()
{
  auto& pass    = *_pass_itm_data;
  auto& subpass = *pass.subpass;

  auto& pages = subpass.pages;

  // TODO: this could be made smaller if we ignored dictionary pages and pages with no
  // repetition data.
  size_t const per_page_decode_buf_size = LEVEL_DECODE_BUF_SIZE * 2 * pass.level_type_size;
  auto const decode_buf_size            = per_page_decode_buf_size * pages.size();
  subpass.level_decode_data =
    rmm::device_buffer(decode_buf_size, _stream, cudf::get_current_device_resource_ref());

  // distribute the buffers
  auto* buf = static_cast<uint8_t*>(subpass.level_decode_data.data());
  for (size_t idx = 0; idx < pages.size(); idx++) {
    auto& p = pages[idx];

    p.lvl_decode_buf[level_type::DEFINITION] = buf;
    buf += (LEVEL_DECODE_BUF_SIZE * pass.level_type_size);
    p.lvl_decode_buf[level_type::REPETITION] = buf;
    buf += (LEVEL_DECODE_BUF_SIZE * pass.level_type_size);
  }
}

std::pair<bool, std::future<void>> reader::impl::read_column_chunks()
{
  auto const& row_groups_info = _pass_itm_data->row_groups;

  auto& raw_page_data = _pass_itm_data->raw_page_data;
  auto& chunks        = _pass_itm_data->chunks;

  // Descriptors for all the chunks that make up the selected columns
  auto const num_input_columns = _input_columns.size();
  auto const num_chunks        = row_groups_info.size() * num_input_columns;

  // Association between each column chunk and its source
  std::vector<size_type> chunk_source_map(num_chunks);

  // Tracker for eventually deallocating compressed and uncompressed data
  raw_page_data = std::vector<rmm::device_buffer>(num_chunks);

  // Keep track of column chunk file offsets
  std::vector<size_t> column_chunk_offsets(num_chunks);

  // Initialize column chunk information
  size_t total_decompressed_size = 0;
  // TODO: make this respect the pass-wide skip_rows/num_rows instead of the file-wide
  // skip_rows/num_rows
  // auto remaining_rows            = num_rows;
  size_type chunk_count = 0;
  for (auto const& rg : row_groups_info) {
    auto const& row_group       = _metadata->get_row_group(rg.index, rg.source_index);
    auto const row_group_source = rg.source_index;

    // generate ColumnChunkDesc objects for everything to be decoded (all input columns)
    for (size_t i = 0; i < num_input_columns; ++i) {
      auto const& col = _input_columns[i];
      // look up metadata
      auto& col_meta = _metadata->get_column_metadata(rg.index, rg.source_index, col.schema_idx);

      column_chunk_offsets[chunk_count] =
        (col_meta.dictionary_page_offset != 0)
          ? std::min(col_meta.data_page_offset, col_meta.dictionary_page_offset)
          : col_meta.data_page_offset;

      // Map each column chunk to its column index and its source index
      chunk_source_map[chunk_count] = row_group_source;

      if (col_meta.codec != Compression::UNCOMPRESSED) {
        total_decompressed_size += col_meta.total_uncompressed_size;
      }

      chunk_count++;
    }
  }

  // Read compressed chunk data to device memory
  return {total_decompressed_size > 0,
          read_column_chunks_async(_sources,
                                   raw_page_data,
                                   chunks,
                                   0,
                                   chunks.size(),
                                   column_chunk_offsets,
                                   chunk_source_map,
                                   _stream)};
}

void reader::impl::read_compressed_data()
{
  auto& pass = *_pass_itm_data;

  // This function should never be called if `num_rows == 0`.
  CUDF_EXPECTS(_pass_itm_data->num_rows > 0, "Number of reading rows must not be zero.");

  auto& chunks = pass.chunks;

  auto const [has_compressed_data, read_chunks_tasks] = read_column_chunks();
  pass.has_compressed_data                            = has_compressed_data;

  read_chunks_tasks.wait();

  // Process dataset chunk pages into output columns
  auto const total_pages = _has_page_index ? count_page_headers_with_pgidx(chunks, _stream)
                                           : count_page_headers(chunks, _stream);
  if (total_pages <= 0) { return; }
  rmm::device_uvector<PageInfo> unsorted_pages(total_pages, _stream);

  // decoding of column/page information
  decode_page_headers(pass, unsorted_pages, _has_page_index, _stream);
  CUDF_EXPECTS(pass.page_offsets.size() - 1 == static_cast<size_t>(_input_columns.size()),
               "Encountered page_offsets / num_columns mismatch");
}

void reader::impl::preprocess_file(read_mode mode)
{
  CUDF_EXPECTS(!_file_preprocessed, "Attempted to preprocess file more than once");

  // if filter is not empty, then create output types as vector and pass for filtering.

  std::vector<data_type> output_dtypes;
  if (_expr_conv.get_converted_expr().has_value()) {
    std::transform(_output_buffers_template.cbegin(),
                   _output_buffers_template.cend(),
                   std::back_inserter(output_dtypes),
                   [](auto const& col) { return col.type; });
  }

  std::tie(_file_itm_data.global_skip_rows,
           _file_itm_data.global_num_rows,
           _file_itm_data.row_groups,
           _file_itm_data.num_rows_per_source,
           _file_itm_data.num_input_row_groups,
           _file_itm_data.surviving_row_groups) =
    _metadata->select_row_groups(_sources,
                                 _options.row_group_indices,
                                 _options.skip_rows,
                                 _options.num_rows,
                                 output_dtypes,
                                 _output_column_schemas,
                                 _expr_conv.get_converted_expr(),
                                 _stream);

  // Inclusive scan the number of rows per source
  if (not _expr_conv.get_converted_expr().has_value() and mode == read_mode::CHUNKED_READ) {
    _file_itm_data.exclusive_sum_num_rows_per_source.resize(
      _file_itm_data.num_rows_per_source.size());
    thrust::inclusive_scan(_file_itm_data.num_rows_per_source.cbegin(),
                           _file_itm_data.num_rows_per_source.cend(),
                           _file_itm_data.exclusive_sum_num_rows_per_source.begin());
  }

  // check for page indexes
  _has_page_index = std::all_of(_file_itm_data.row_groups.cbegin(),
                                _file_itm_data.row_groups.cend(),
                                [](auto const& row_group) { return row_group.has_page_index(); });

  if (_file_itm_data.global_num_rows > 0 && not _file_itm_data.row_groups.empty() &&
      not _input_columns.empty()) {
    // fills in chunk information without physically loading or decompressing
    // the associated data
    create_global_chunk_info();

    // compute schedule of input reads.
    compute_input_passes();
  }

#if defined(PARQUET_CHUNK_LOGGING)
  printf("==============================================\n");
  setlocale(LC_NUMERIC, "");
  printf("File: skip_rows(%'lu), num_rows(%'lu), input_read_limit(%'lu), output_read_limit(%'lu)\n",
         _file_itm_data.global_skip_rows,
         _file_itm_data.global_num_rows,
         _input_pass_read_limit,
         _output_chunk_read_limit);
  printf("# Row groups: %'lu\n", _file_itm_data.row_groups.size());
  printf("# Input passes: %'lu\n", _file_itm_data.num_passes());
  printf("# Input columns: %'lu\n", _input_columns.size());
  for (size_t idx = 0; idx < _input_columns.size(); idx++) {
    auto const& schema = _metadata->get_schema(_input_columns[idx].schema_idx);
    auto const type_id = to_type_id(schema, _strings_to_categorical, _options.timestamp_type.id());
    printf("\tC(%'lu, %s): %s\n",
           idx,
           _input_columns[idx].name.c_str(),
           cudf::type_to_name(cudf::data_type{type_id}).c_str());
  }
  printf("# Output columns: %'lu\n", _output_buffers.size());
  for (size_t idx = 0; idx < _output_buffers.size(); idx++) {
    printf("\tC(%'lu): %s\n", idx, cudf::io::detail::type_to_name(_output_buffers[idx]).c_str());
  }
#endif

  _file_preprocessed = true;
}

void reader::impl::generate_list_column_row_counts(is_estimate_row_counts is_estimate_row_counts)
{
  auto& pass = *_pass_itm_data;

  // Computes:
  // Estimated PageInfo::chunk_row (the chunk-relative row index) and PageInfo::num_rows (number of
  // rows in this page) for all pages in the pass. The start_row field in ColumnChunkDesc is the
  // absolute row index for the whole file. chunk_row in PageInfo is relative to the beginning of
  // the chunk. so in the kernels, chunk.start_row + page.chunk_row gives us the absolute row index
  if (is_estimate_row_counts == is_estimate_row_counts::YES) {
    thrust::for_each(rmm::exec_policy(_stream),
                     pass.pages.d_begin(),
                     pass.pages.d_end(),
                     set_list_row_count_estimate{pass.chunks});
    auto key_input  = thrust::make_transform_iterator(pass.pages.d_begin(), get_page_chunk_idx{});
    auto page_input = thrust::make_transform_iterator(pass.pages.d_begin(), get_page_num_rows{});
    thrust::exclusive_scan_by_key(rmm::exec_policy_nosync(_stream),
                                  key_input,
                                  key_input + pass.pages.size(),
                                  page_input,
                                  chunk_row_output_iter{pass.pages.device_ptr()});

    // To compensate for the list row size estimates, force the row count on the last page for each
    // column chunk (each rowgroup) such that it ends on the real known row count. this is so that
    // as we march through the subpasses, we will find that every column cleanly ends up the
    // expected row count at the row group boundary and our split computations work correctly.
    auto iter = thrust::make_counting_iterator(0);
    thrust::for_each(rmm::exec_policy_nosync(_stream),
                     iter,
                     iter + pass.pages.size(),
                     set_final_row_count{pass.pages, pass.chunks});
  } else {
    // If column indexes are available, we don't need to estimate PageInfo::num_rows for lists and
    // can instead translate known PageInfo::chunk_row to PageInfo::num_rows
    thrust::for_each(rmm::exec_policy_nosync(_stream),
                     thrust::counting_iterator<size_t>(0),
                     thrust::counting_iterator(pass.pages.size()),
                     compute_page_num_rows_from_chunk_rows{pass.pages, pass.chunks});
  }

  pass.chunks.device_to_host_async(_stream);
  pass.pages.device_to_host_async(_stream);
  _stream.synchronize();
}

void reader::impl::preprocess_subpass_pages(read_mode mode, size_t chunk_read_limit)
{
  auto& pass    = *_pass_itm_data;
  auto& subpass = *pass.subpass;

  // iterate over all input columns and determine if they contain lists.
  // TODO: we could do this once at the file level instead of every time we get in here. the set of
  // columns we are processing does not change over multiple passes/subpasses/output chunks.
  bool has_lists = false;
  for (const auto& input_col : _input_columns) {
    size_t const max_depth = input_col.nesting_depth();

    auto* cols = &_output_buffers;
    for (size_t l_idx = 0; l_idx < max_depth; l_idx++) {
      auto& out_buf = (*cols)[input_col.nesting[l_idx]];
      cols          = &out_buf.children;

      // if this has a list parent, we have to get column sizes from the
      // data computed during compute_page_sizes
      if (out_buf.user_data & PARQUET_COLUMN_BUFFER_FLAG_HAS_LIST_PARENT) {
        has_lists = true;
        break;
      }
    }
    if (has_lists) { break; }
  }

  // in some cases we will need to do further preprocessing of pages.
  // - if we have lists, the num_rows field in PageInfo will be incorrect coming out of the file
  // - if we are doing a chunked read, we need to compute the size of all string data
  if (has_lists || chunk_read_limit > 0) {
    // computes:
    // PageNestingInfo::num_rows for each page. the true number of rows (taking repetition into
    // account), not just the number of values. PageNestingInfo::size for each level of nesting, for
    // each page.
    //
    // we will be applying a later "trim" pass if skip_rows/num_rows is being used, which can happen
    // if:
    // - user has passed custom row bounds
    // - we will be doing a chunked read
    compute_page_sizes(subpass.pages,
                       pass.chunks,
                       0,  // 0-max size_t. process all possible rows
                       std::numeric_limits<size_t>::max(),
                       true,                  // compute num_rows
                       chunk_read_limit > 0,  // compute string sizes
                       _pass_itm_data->level_type_size,
                       _stream);
  }

  auto iter = thrust::make_counting_iterator(0);

  // copy our now-correct row counts  back to the base pages stored in the pass.
  // only need to do this if we are not processing the whole pass in one subpass
  if (!subpass.single_subpass) {
    thrust::for_each(rmm::exec_policy_nosync(_stream),
                     iter,
                     iter + subpass.pages.size(),
                     update_pass_num_rows{pass.pages, subpass.pages, subpass.page_src_index});
  }

  // computes:
  // PageInfo::chunk_row (the chunk-relative row index) for all pages in the pass. The start_row
  // field in ColumnChunkDesc is the absolute row index for the whole file. chunk_row in PageInfo is
  // relative to the beginning of the chunk. so in the kernels, chunk.start_row + page.chunk_row
  // gives us the absolute row index
  auto key_input  = thrust::make_transform_iterator(pass.pages.d_begin(), get_page_chunk_idx{});
  auto page_input = thrust::make_transform_iterator(pass.pages.d_begin(), get_page_num_rows{});
  thrust::exclusive_scan_by_key(rmm::exec_policy_nosync(_stream),
                                key_input,
                                key_input + pass.pages.size(),
                                page_input,
                                chunk_row_output_iter{pass.pages.device_ptr()});

  // copy chunk_row into the subpass pages
  // only need to do this if we are not processing the whole pass in one subpass
  if (!subpass.single_subpass) {
    thrust::for_each(rmm::exec_policy_nosync(_stream),
                     iter,
                     iter + subpass.pages.size(),
                     update_subpass_chunk_row{pass.pages, subpass.pages, subpass.page_src_index});
  }

  // retrieve pages back
  pass.pages.device_to_host_async(_stream);
  if (!subpass.single_subpass) { subpass.pages.device_to_host_async(_stream); }
  _stream.synchronize();

  // at this point we have an accurate row count so we can compute how many rows we will actually be
  // able to decode for this pass. we will have selected a set of pages for each column in the
  // row group, but not every page will have the same number of rows. so, we can only read as many
  // rows as the smallest batch (by column) we have decompressed.
  size_t first_page_index = 0;
  size_t max_row          = std::numeric_limits<size_t>::max();
  auto const last_pass_row =
    _file_itm_data.input_pass_start_row_count[_file_itm_data._current_input_pass + 1];
  // for each column
  for (size_t idx = 0; idx < subpass.column_page_count.size(); idx++) {
    // compute max row for this column in the subpass
    auto const& last_page  = subpass.pages[first_page_index + (subpass.column_page_count[idx] - 1)];
    auto const& last_chunk = pass.chunks[last_page.chunk_idx];
    auto max_col_row       = static_cast<size_t>(last_chunk.start_row) +
                       static_cast<size_t>(last_page.chunk_row) +
                       static_cast<size_t>(last_page.num_rows);

    // special case.  list rows can span page boundaries, but we can't tell if that is happening
    // here because we have not yet decoded the pages. the very last row starting in the page may
    // not terminate in the page. to handle this, only decode up to the second to last row in the
    // subpass since we know that will safely completed.
    bool const is_list = last_chunk.max_level[level_type::REPETITION] > 0;
    // corner case: only decode up to the second-to-last row, except if this is the last page in the
    // entire pass. this handles the case where we only have 1 chunk, 1 page, and potentially even
    // just 1 row.
    if (is_list && max_col_row < last_pass_row) {
      // compute min row for this column in the subpass
      auto const& first_page  = subpass.pages[first_page_index];
      auto const& first_chunk = pass.chunks[first_page.chunk_idx];
      auto const min_col_row =
        static_cast<size_t>(first_chunk.start_row) + static_cast<size_t>(first_page.chunk_row);

      // must have at least 2 rows in the subpass.
      CUDF_EXPECTS((max_col_row - min_col_row) > 1, "Unexpected short subpass");
      max_col_row--;
    }

    max_row = std::min<size_t>(max_row, max_col_row);

    first_page_index += subpass.column_page_count[idx];
  }
  subpass.skip_rows   = pass.skip_rows + pass.processed_rows;
  auto const pass_end = pass.skip_rows + pass.num_rows;
  max_row             = std::min<size_t>(max_row, pass_end);
  CUDF_EXPECTS(max_row > subpass.skip_rows, "Unexpected short subpass", std::underflow_error);
  subpass.num_rows = max_row - subpass.skip_rows;

  // now split up the output into chunks as necessary
  compute_output_chunks_for_subpass();
}

void reader::impl::allocate_columns(read_mode mode, size_t skip_rows, size_t num_rows)
{
  auto& pass    = *_pass_itm_data;
  auto& subpass = *pass.subpass;

  // Should not reach here if there is no page data.
  CUDF_EXPECTS(subpass.pages.size() > 0, "There are no pages present in the subpass");

  // computes:
  // PageNestingInfo::batch_size for each level of nesting, for each page, taking row bounds into
  // account. PageInfo::skipped_values, which tells us where to start decoding in the input to
  // respect the user bounds. It is only necessary to do this second pass if uses_custom_row_bounds
  // is set (if the user has specified artificial bounds).
  if (uses_custom_row_bounds(mode)) {
    compute_page_sizes(subpass.pages,
                       pass.chunks,
                       skip_rows,
                       num_rows,
                       false,  // num_rows is already computed
                       false,  // no need to compute string sizes
                       pass.level_type_size,
                       _stream);
  }

  // iterate over all input columns and allocate any associated output
  // buffers if they are not part of a list hierarchy. mark down
  // if we have any list columns that need further processing.
  bool has_lists = false;
  // Casting to std::byte since data buffer pointer is void *
  std::vector<cudf::device_span<cuda::std::byte>> memset_bufs;
  // Validity Buffer is a uint32_t pointer
  std::vector<cudf::device_span<cudf::bitmask_type>> nullmask_bufs;

  for (const auto& input_col : _input_columns) {
    size_t const max_depth = input_col.nesting_depth();

    auto* cols = &_output_buffers;
    for (size_t l_idx = 0; l_idx < max_depth; l_idx++) {
      auto& out_buf = (*cols)[input_col.nesting[l_idx]];
      cols          = &out_buf.children;

      // if this has a list parent, we have to get column sizes from the
      // data computed during compute_page_sizes
      if (out_buf.user_data & PARQUET_COLUMN_BUFFER_FLAG_HAS_LIST_PARENT) {
        has_lists = true;
      }
      // if we haven't already processed this column because it is part of a struct hierarchy
      else if (out_buf.size == 0) {
        // add 1 for the offset if this is a list column
        // we're going to start null mask as all valid and then turn bits off if necessary
        auto const out_buf_size =
          out_buf.type.id() == type_id::LIST && l_idx < max_depth ? num_rows + 1 : num_rows;
        CUDF_EXPECTS(out_buf_size <= std::numeric_limits<cudf::size_type>::max(),
                     "Number of rows exceeds cudf's column size limit",
                     std::overflow_error);
        out_buf.create_with_mask(
          out_buf_size, cudf::mask_state::UNINITIALIZED, false, _stream, _mr);
        memset_bufs.emplace_back(static_cast<cuda::std::byte*>(out_buf.data()),
                                 out_buf.data_size());
        nullmask_bufs.emplace_back(
          out_buf.null_mask(),
          cudf::util::round_up_safe(out_buf.null_mask_size(), sizeof(cudf::bitmask_type)) /
            sizeof(cudf::bitmask_type));
      }
    }
  }
  // compute output column sizes by examining the pages of the -input- columns
  if (has_lists) {
    auto h_cols_info =
      cudf::detail::make_empty_host_vector<input_col_info>(_input_columns.size(), _stream);
    std::transform(_input_columns.cbegin(),
                   _input_columns.cend(),
                   std::back_inserter(h_cols_info),
                   [](auto& col) -> input_col_info {
                     return {col.schema_idx, static_cast<size_type>(col.nesting_depth())};
                   });

    auto const max_depth =
      (*std::max_element(h_cols_info.cbegin(),
                         h_cols_info.cend(),
                         [](auto& l, auto& r) { return l.nesting_depth < r.nesting_depth; }))
        .nesting_depth;

    auto const d_cols_info = cudf::detail::make_device_uvector_async(
      h_cols_info, _stream, cudf::get_current_device_resource_ref());

    // Vector to store page sizes for each column at each depth
    cudf::detail::hostdevice_vector<size_t> sizes{_input_columns.size() * max_depth, _stream};

    // Total number of keys to process
    auto const num_keys = _input_columns.size() * max_depth * subpass.pages.size();

    // Maximum 1 billion keys processed per iteration
    auto constexpr max_keys_per_iter =
      static_cast<size_t>(std::numeric_limits<size_type>::max() / 2);

    // Number of keys for per each column
    auto const num_keys_per_col = max_depth * subpass.pages.size();

    // The largest multiple of `num_keys_per_col` that is <= `num_keys`
    auto const num_keys_per_iter =
      num_keys <= max_keys_per_iter
        ? num_keys
        : num_keys_per_col * std::max<size_t>(1, max_keys_per_iter / num_keys_per_col);

    // Size iterator. Indexes pages by sorted order
    rmm::device_uvector<size_type> size_input{num_keys_per_iter, _stream};

    // To keep track of the starting key of an iteration
    size_t key_start = 0;
    // Loop until all keys are processed
    while (key_start < num_keys) {
      // Number of keys processed in this iteration
      auto const num_keys_this_iter = std::min<size_t>(num_keys_per_iter, num_keys - key_start);
      thrust::transform(
        rmm::exec_policy_nosync(_stream),
        thrust::make_counting_iterator<size_t>(key_start),
        thrust::make_counting_iterator<size_t>(key_start + num_keys_this_iter),
        size_input.begin(),
        get_page_nesting_size{
          d_cols_info.data(), max_depth, subpass.pages.size(), subpass.pages.device_begin()});

      // Manually create a size_t `key_start` compatible counting_transform_iterator.
      auto const reduction_keys =
        thrust::make_transform_iterator(thrust::make_counting_iterator<std::size_t>(key_start),
                                        get_reduction_key{subpass.pages.size()});

      // Find the size of each column
      thrust::reduce_by_key(rmm::exec_policy_nosync(_stream),
                            reduction_keys,
                            reduction_keys + num_keys_this_iter,
                            size_input.cbegin(),
                            thrust::make_discard_iterator(),
                            sizes.d_begin() + (key_start / subpass.pages.size()));

      // For nested hierarchies, compute per-page start offset
      thrust::exclusive_scan_by_key(rmm::exec_policy_nosync(_stream),
                                    reduction_keys,
                                    reduction_keys + num_keys_this_iter,
                                    size_input.cbegin(),
                                    start_offset_output_iterator{subpass.pages.device_begin(),
                                                                 key_start,
                                                                 d_cols_info.data(),
                                                                 max_depth,
                                                                 subpass.pages.size()});
      // Increment the key_start
      key_start += num_keys_this_iter;
    }

    sizes.device_to_host(_stream);
    for (size_type idx = 0; idx < static_cast<size_type>(_input_columns.size()); idx++) {
      auto const& input_col = _input_columns[idx];
      auto* cols            = &_output_buffers;
      for (size_type l_idx = 0; l_idx < static_cast<size_type>(input_col.nesting_depth());
           l_idx++) {
        auto& out_buf = (*cols)[input_col.nesting[l_idx]];
        cols          = &out_buf.children;
        // if this buffer is part of a list hierarchy, we need to determine it's
        // final size and allocate it here.
        //
        // for struct columns, higher levels of the output columns are shared between input
        // columns. so don't compute any given level more than once.
        if ((out_buf.user_data & PARQUET_COLUMN_BUFFER_FLAG_HAS_LIST_PARENT) && out_buf.size == 0) {
          auto buffer_size = sizes[(idx * max_depth) + l_idx];
          // if this is a list column add 1 for non-leaf levels for the terminating offset
          if (out_buf.type.id() == type_id::LIST && l_idx < max_depth) { buffer_size++; }
          CUDF_EXPECTS(buffer_size <= std::numeric_limits<cudf::size_type>::max(),
                       "Number of list column rows exceeds cudf's column size limit",
                       std::overflow_error);
          // allocate
          // we're going to start null mask as all valid and then turn bits off if necessary
          out_buf.create_with_mask(
            buffer_size, cudf::mask_state::UNINITIALIZED, false, _stream, _mr);
          memset_bufs.emplace_back(static_cast<cuda::std::byte*>(out_buf.data()),
                                   out_buf.data_size());
          nullmask_bufs.emplace_back(
            out_buf.null_mask(),
            cudf::util::round_up_safe(out_buf.null_mask_size(), sizeof(cudf::bitmask_type)) /
              sizeof(cudf::bitmask_type));
        }
      }
    }
  }

  cudf::detail::batched_memset<cuda::std::byte>(
    memset_bufs, static_cast<cuda::std::byte>(0), _stream);
  // Need to set null mask bufs to all high bits
  cudf::detail::batched_memset<cudf::bitmask_type>(
    nullmask_bufs, std::numeric_limits<cudf::bitmask_type>::max(), _stream);
}

cudf::detail::host_vector<size_t> reader::impl::calculate_page_string_offsets()
{
  auto& pass    = *_pass_itm_data;
  auto& subpass = *pass.subpass;

  auto page_keys = make_page_key_iterator(subpass.pages);

  rmm::device_uvector<size_t> d_col_sizes(_input_columns.size(), _stream);

  // use page_index to fetch page string sizes in the proper order
  auto val_iter = thrust::make_transform_iterator(subpass.pages.device_begin(),
                                                  page_to_string_size{pass.chunks.d_begin()});

  // do scan by key to calculate string offsets for each page
  thrust::exclusive_scan_by_key(rmm::exec_policy_nosync(_stream),
                                page_keys,
                                page_keys + subpass.pages.size(),
                                val_iter,
                                page_offset_output_iter{subpass.pages.device_ptr()});

  // now sum up page sizes
  rmm::device_uvector<int> reduce_keys(d_col_sizes.size(), _stream);
  thrust::reduce_by_key(rmm::exec_policy_nosync(_stream),
                        page_keys,
                        page_keys + subpass.pages.size(),
                        val_iter,
                        reduce_keys.begin(),
                        d_col_sizes.begin());

  return cudf::detail::make_host_vector(d_col_sizes, _stream);
}

}  // namespace cudf::io::parquet::detail
