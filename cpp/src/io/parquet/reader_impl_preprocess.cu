#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2022, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "reader_impl.hpp"

#include <io/comp/nvcomp_adapter.hpp>
#include <io/utilities/config_utils.hpp>
#include <io/utilities/time_utils.cuh>

#include <cudf/detail/iterator.cuh>
#include <cudf/detail/utilities/integer_utils.hpp>
#include <cudf/detail/utilities/vector_factories.hpp>

#include <rmm/exec_policy.hpp>

#include <thrust/binary_search.h>
#include <thrust/fill.h>
#include <thrust/functional.h>
#include <thrust/iterator/constant_iterator.h>
#include <thrust/iterator/discard_iterator.h>
#include <thrust/iterator/iterator_categories.h>
#include <thrust/iterator/transform_iterator.h>
#include <thrust/logical.h>
#include <thrust/reduce.h>
#include <thrust/scan.h>
#include <thrust/sequence.h>
#include <thrust/sort.h>
#include <thrust/transform.h>
#include <thrust/unique.h>

#include <numeric>

namespace cudf::io::detail::parquet {
namespace {

/**
 * @brief Generate depth remappings for repetition and definition levels.
 *
 * When dealing with columns that contain lists, we must examine incoming
 * repetition and definition level pairs to determine what range of output nesting
 * is indicated when adding new values.  This function generates the mappings of
 * the R/D levels to those start/end bounds
 *
 * @param remap Maps column schema index to the R/D remapping vectors for that column
 * @param src_col_schema The column schema to generate the new mapping for
 * @param md File metadata information
 */
void generate_depth_remappings(std::map<int, std::pair<std::vector<int>, std::vector<int>>>& remap,
                               int src_col_schema,
                               aggregate_reader_metadata const& md)
{
  // already generated for this level
  if (remap.find(src_col_schema) != remap.end()) { return; }
  auto schema   = md.get_schema(src_col_schema);
  int max_depth = md.get_output_nesting_depth(src_col_schema);

  CUDF_EXPECTS(remap.find(src_col_schema) == remap.end(),
               "Attempting to remap a schema more than once");
  auto inserted =
    remap.insert(std::pair<int, std::pair<std::vector<int>, std::vector<int>>>{src_col_schema, {}});
  auto& depth_remap = inserted.first->second;

  std::vector<int>& rep_depth_remap = (depth_remap.first);
  rep_depth_remap.resize(schema.max_repetition_level + 1);
  std::vector<int>& def_depth_remap = (depth_remap.second);
  def_depth_remap.resize(schema.max_definition_level + 1);

  // the key:
  // for incoming level values  R/D
  // add values starting at the shallowest nesting level X has repetition level R
  // until you reach the deepest nesting level Y that corresponds to the repetition level R1
  // held by the nesting level that has definition level D
  //
  // Example: a 3 level struct with a list at the bottom
  //
  //                     R / D   Depth
  // level0              0 / 1     0
  //   level1            0 / 2     1
  //     level2          0 / 3     2
  //       list          0 / 3     3
  //         element     1 / 4     4
  //
  // incoming R/D : 0, 0  -> add values from depth 0 to 3   (def level 0 always maps to depth 0)
  // incoming R/D : 0, 1  -> add values from depth 0 to 3
  // incoming R/D : 0, 2  -> add values from depth 0 to 3
  // incoming R/D : 1, 4  -> add values from depth 4 to 4
  //
  // Note : the -validity- of values is simply checked by comparing the incoming D value against the
  // D value of the given nesting level (incoming D >= the D for the nesting level == valid,
  // otherwise NULL).  The tricky part is determining what nesting levels to add values at.
  //
  // For schemas with no repetition level (no lists), X is always 0 and Y is always max nesting
  // depth.
  //

  // compute "X" from above
  for (int s_idx = schema.max_repetition_level; s_idx >= 0; s_idx--) {
    auto find_shallowest = [&](int r) {
      int shallowest = -1;
      int cur_depth  = max_depth - 1;
      int schema_idx = src_col_schema;
      while (schema_idx > 0) {
        auto cur_schema = md.get_schema(schema_idx);
        if (cur_schema.max_repetition_level == r) {
          // if this is a repeated field, map it one level deeper
          shallowest = cur_schema.is_stub() ? cur_depth + 1 : cur_depth;
        }
        // if it's one-level encoding list
        else if (cur_schema.is_one_level_list()) {
          shallowest = cur_depth - 1;
        }
        if (!cur_schema.is_stub()) { cur_depth--; }
        schema_idx = cur_schema.parent_idx;
      }
      return shallowest;
    };
    rep_depth_remap[s_idx] = find_shallowest(s_idx);
  }

  // compute "Y" from above
  for (int s_idx = schema.max_definition_level; s_idx >= 0; s_idx--) {
    auto find_deepest = [&](int d) {
      SchemaElement prev_schema;
      int schema_idx = src_col_schema;
      int r1         = 0;
      while (schema_idx > 0) {
        SchemaElement cur_schema = md.get_schema(schema_idx);
        if (cur_schema.max_definition_level == d) {
          // if this is a repeated field, map it one level deeper
          r1 = cur_schema.is_stub() ? prev_schema.max_repetition_level
                                    : cur_schema.max_repetition_level;
          break;
        }
        prev_schema = cur_schema;
        schema_idx  = cur_schema.parent_idx;
      }

      // we now know R1 from above. return the deepest nesting level that has the
      // same repetition level
      schema_idx = src_col_schema;
      int depth  = max_depth - 1;
      while (schema_idx > 0) {
        SchemaElement cur_schema = md.get_schema(schema_idx);
        if (cur_schema.max_repetition_level == r1) {
          // if this is a repeated field, map it one level deeper
          depth = cur_schema.is_stub() ? depth + 1 : depth;
          break;
        }
        if (!cur_schema.is_stub()) { depth--; }
        prev_schema = cur_schema;
        schema_idx  = cur_schema.parent_idx;
      }
      return depth;
    };
    def_depth_remap[s_idx] = find_deepest(s_idx);
  }
}

/**
 * @brief Return the required number of bits to store a value.
 */
template <typename T = uint8_t>
[[nodiscard]] T required_bits(uint32_t max_level)
{
  return static_cast<T>(CompactProtocolReader::NumRequiredBits(max_level));
}

/**
 * @brief Converts cuDF units to Parquet units.
 *
 * @return A tuple of Parquet type width, Parquet clock rate and Parquet decimal type.
 */
[[nodiscard]] std::tuple<int32_t, int32_t, int8_t> conversion_info(type_id column_type_id,
                                                                   type_id timestamp_type_id,
                                                                   parquet::Type physical,
                                                                   int8_t converted,
                                                                   int32_t length)
{
  int32_t type_width = (physical == parquet::FIXED_LEN_BYTE_ARRAY) ? length : 0;
  int32_t clock_rate = 0;
  if (column_type_id == type_id::INT8 or column_type_id == type_id::UINT8) {
    type_width = 1;  // I32 -> I8
  } else if (column_type_id == type_id::INT16 or column_type_id == type_id::UINT16) {
    type_width = 2;  // I32 -> I16
  } else if (column_type_id == type_id::INT32) {
    type_width = 4;  // str -> hash32
  } else if (is_chrono(data_type{column_type_id})) {
    clock_rate = to_clockrate(timestamp_type_id);
  }

  int8_t converted_type = converted;
  if (converted_type == parquet::DECIMAL && column_type_id != type_id::FLOAT64 &&
      not cudf::is_fixed_point(data_type{column_type_id})) {
    converted_type = parquet::UNKNOWN;  // Not converting to float64 or decimal
  }
  return std::make_tuple(type_width, clock_rate, converted_type);
}

/**
 * @brief Reads compressed page data to device memory.
 *
 * @param sources Dataset sources
 * @param page_data Buffers to hold compressed page data for each chunk
 * @param chunks List of column chunk descriptors
 * @param begin_chunk Index of first column chunk to read
 * @param end_chunk Index after the last column chunk to read
 * @param column_chunk_offsets File offset for all chunks
 * @param chunk_source_map Association between each column chunk and its source
 * @param stream CUDA stream used for device memory operations and kernel launches
 *
 * @return A future object for reading synchronization
 */
[[nodiscard]] std::future<void> read_column_chunks_async(
  std::vector<std::unique_ptr<datasource>> const& sources,
  std::vector<std::unique_ptr<datasource::buffer>>& page_data,
  hostdevice_vector<gpu::ColumnChunkDesc>& chunks,
  size_t begin_chunk,
  size_t end_chunk,
  const std::vector<size_t>& column_chunk_offsets,
  std::vector<size_type> const& chunk_source_map,
  rmm::cuda_stream_view stream)
{
  // Transfer chunk data, coalescing adjacent chunks
  std::vector<std::future<size_t>> read_tasks;
  for (size_t chunk = begin_chunk; chunk < end_chunk;) {
    const size_t io_offset   = column_chunk_offsets[chunk];
    size_t io_size           = chunks[chunk].compressed_size;
    size_t next_chunk        = chunk + 1;
    const bool is_compressed = (chunks[chunk].codec != parquet::Compression::UNCOMPRESSED);
    while (next_chunk < end_chunk) {
      const size_t next_offset = column_chunk_offsets[next_chunk];
      const bool is_next_compressed =
        (chunks[next_chunk].codec != parquet::Compression::UNCOMPRESSED);
      if (next_offset != io_offset + io_size || is_next_compressed != is_compressed) {
        // Can't merge if not contiguous or mixing compressed and uncompressed
        // Not coalescing uncompressed with compressed chunks is so that compressed buffers can be
        // freed earlier (immediately after decompression stage) to limit peak memory requirements
        break;
      }
      io_size += chunks[next_chunk].compressed_size;
      next_chunk++;
    }
    if (io_size != 0) {
      auto& source = sources[chunk_source_map[chunk]];
      if (source->is_device_read_preferred(io_size)) {
        auto buffer        = rmm::device_buffer(io_size, stream);
        auto fut_read_size = source->device_read_async(
          io_offset, io_size, static_cast<uint8_t*>(buffer.data()), stream);
        read_tasks.emplace_back(std::move(fut_read_size));
        page_data[chunk] = datasource::buffer::create(std::move(buffer));
      } else {
        auto const buffer = source->host_read(io_offset, io_size);
        page_data[chunk] =
          datasource::buffer::create(rmm::device_buffer(buffer->data(), buffer->size(), stream));
      }
      auto d_compdata = page_data[chunk]->data();
      do {
        chunks[chunk].compressed_data = d_compdata;
        d_compdata += chunks[chunk].compressed_size;
      } while (++chunk != next_chunk);
    } else {
      chunk = next_chunk;
    }
  }
  auto sync_fn = [](decltype(read_tasks) read_tasks) {
    for (auto& task : read_tasks) {
      task.wait();
    }
  };
  return std::async(std::launch::deferred, sync_fn, std::move(read_tasks));
}

/**
 * @brief Return the number of total pages from the given column chunks.
 *
 * @param chunks List of column chunk descriptors
 * @param stream CUDA stream used for device memory operations and kernel launches
 *
 * @return The total number of pages
 */
[[nodiscard]] size_t count_page_headers(hostdevice_vector<gpu::ColumnChunkDesc>& chunks,
                                        rmm::cuda_stream_view stream)
{
  size_t total_pages = 0;

  chunks.host_to_device(stream);
  gpu::DecodePageHeaders(chunks.device_ptr(), chunks.size(), stream);
  chunks.device_to_host(stream, true);

  for (size_t c = 0; c < chunks.size(); c++) {
    total_pages += chunks[c].num_data_pages + chunks[c].num_dict_pages;
  }

  return total_pages;
}

/**
 * @brief Decode the page information from the given column chunks.
 *
 * @param chunks List of column chunk descriptors
 * @param pages List of page information
 * @param stream CUDA stream used for device memory operations and kernel launches
 */
void decode_page_headers(hostdevice_vector<gpu::ColumnChunkDesc>& chunks,
                         hostdevice_vector<gpu::PageInfo>& pages,
                         rmm::cuda_stream_view stream)
{
  // IMPORTANT : if you change how pages are stored within a chunk (dist pages, then data pages),
  // please update preprocess_nested_columns to reflect this.
  for (size_t c = 0, page_count = 0; c < chunks.size(); c++) {
    chunks[c].max_num_pages = chunks[c].num_data_pages + chunks[c].num_dict_pages;
    chunks[c].page_info     = pages.device_ptr(page_count);
    page_count += chunks[c].max_num_pages;
  }

  chunks.host_to_device(stream);
  gpu::DecodePageHeaders(chunks.device_ptr(), chunks.size(), stream);
  pages.device_to_host(stream, true);
}

/**
 * @brief Decompresses the page data, at page granularity.
 *
 * @param chunks List of column chunk descriptors
 * @param pages List of page information
 * @param stream CUDA stream used for device memory operations and kernel launches
 *
 * @return Device buffer to decompressed page data
 */
[[nodiscard]] rmm::device_buffer decompress_page_data(
  hostdevice_vector<gpu::ColumnChunkDesc>& chunks,
  hostdevice_vector<gpu::PageInfo>& pages,
  rmm::cuda_stream_view stream)
{
  auto for_each_codec_page = [&](parquet::Compression codec, const std::function<void(size_t)>& f) {
    for (size_t c = 0, page_count = 0; c < chunks.size(); c++) {
      const auto page_stride = chunks[c].max_num_pages;
      if (chunks[c].codec == codec) {
        for (int k = 0; k < page_stride; k++) {
          f(page_count + k);
        }
      }
      page_count += page_stride;
    }
  };

  // Brotli scratch memory for decompressing
  rmm::device_buffer debrotli_scratch;

  // Count the exact number of compressed pages
  size_t num_comp_pages    = 0;
  size_t total_decomp_size = 0;

  struct codec_stats {
    parquet::Compression compression_type = UNCOMPRESSED;
    size_t num_pages                      = 0;
    int32_t max_decompressed_size         = 0;
    size_t total_decomp_size              = 0;
  };

  std::array codecs{codec_stats{parquet::GZIP},
                    codec_stats{parquet::SNAPPY},
                    codec_stats{parquet::BROTLI},
                    codec_stats{parquet::ZSTD}};

  auto is_codec_supported = [&codecs](int8_t codec) {
    if (codec == parquet::UNCOMPRESSED) return true;
    return std::find_if(codecs.begin(), codecs.end(), [codec](auto& cstats) {
             return codec == cstats.compression_type;
           }) != codecs.end();
  };
  CUDF_EXPECTS(std::all_of(chunks.begin(),
                           chunks.end(),
                           [&is_codec_supported](auto const& chunk) {
                             return is_codec_supported(chunk.codec);
                           }),
               "Unsupported compression type");

  for (auto& codec : codecs) {
    for_each_codec_page(codec.compression_type, [&](size_t page) {
      auto page_uncomp_size = pages[page].uncompressed_page_size;
      total_decomp_size += page_uncomp_size;
      codec.total_decomp_size += page_uncomp_size;
      codec.max_decompressed_size = std::max(codec.max_decompressed_size, page_uncomp_size);
      codec.num_pages++;
      num_comp_pages++;
    });
    if (codec.compression_type == parquet::BROTLI && codec.num_pages > 0) {
      debrotli_scratch.resize(get_gpu_debrotli_scratch_size(codec.num_pages), stream);
    }
  }

  // Dispatch batches of pages to decompress for each codec
  rmm::device_buffer decomp_pages(total_decomp_size, stream);

  std::vector<device_span<uint8_t const>> comp_in;
  comp_in.reserve(num_comp_pages);
  std::vector<device_span<uint8_t>> comp_out;
  comp_out.reserve(num_comp_pages);

  // vectors to save v2 def and rep level data, if any
  std::vector<device_span<uint8_t const>> copy_in;
  copy_in.reserve(num_comp_pages);
  std::vector<device_span<uint8_t>> copy_out;
  copy_out.reserve(num_comp_pages);

  rmm::device_uvector<compression_result> comp_res(num_comp_pages, stream);
  thrust::fill(rmm::exec_policy(stream),
               comp_res.begin(),
               comp_res.end(),
               compression_result{0, compression_status::FAILURE});

  size_t decomp_offset = 0;
  int32_t start_pos    = 0;
  for (const auto& codec : codecs) {
    if (codec.num_pages == 0) { continue; }

    for_each_codec_page(codec.compression_type, [&](size_t page_idx) {
      auto const dst_base = static_cast<uint8_t*>(decomp_pages.data()) + decomp_offset;
      auto& page          = pages[page_idx];
      // offset will only be non-zero for V2 pages
      auto const offset = page.def_lvl_bytes + page.rep_lvl_bytes;
      // for V2 need to copy def and rep level info into place, and then offset the
      // input and output buffers. otherwise we'd have to keep both the compressed
      // and decompressed data.
      if (offset != 0) {
        copy_in.emplace_back(page.page_data, offset);
        copy_out.emplace_back(dst_base, offset);
      }
      comp_in.emplace_back(page.page_data + offset,
                           static_cast<size_t>(page.compressed_page_size - offset));
      comp_out.emplace_back(dst_base + offset,
                            static_cast<size_t>(page.uncompressed_page_size - offset));
      page.page_data = dst_base;
      decomp_offset += page.uncompressed_page_size;
    });

    host_span<device_span<uint8_t const> const> comp_in_view{comp_in.data() + start_pos,
                                                             codec.num_pages};
    auto const d_comp_in = cudf::detail::make_device_uvector_async(comp_in_view, stream);
    host_span<device_span<uint8_t> const> comp_out_view(comp_out.data() + start_pos,
                                                        codec.num_pages);
    auto const d_comp_out = cudf::detail::make_device_uvector_async(comp_out_view, stream);
    device_span<compression_result> d_comp_res_view(comp_res.data() + start_pos, codec.num_pages);

    switch (codec.compression_type) {
      case parquet::GZIP:
        gpuinflate(d_comp_in, d_comp_out, d_comp_res_view, gzip_header_included::YES, stream);
        break;
      case parquet::SNAPPY:
        if (nvcomp_integration::is_stable_enabled()) {
          nvcomp::batched_decompress(nvcomp::compression_type::SNAPPY,
                                     d_comp_in,
                                     d_comp_out,
                                     d_comp_res_view,
                                     codec.max_decompressed_size,
                                     codec.total_decomp_size,
                                     stream);
        } else {
          gpu_unsnap(d_comp_in, d_comp_out, d_comp_res_view, stream);
        }
        break;
      case parquet::ZSTD:
        nvcomp::batched_decompress(nvcomp::compression_type::ZSTD,
                                   d_comp_in,
                                   d_comp_out,
                                   d_comp_res_view,
                                   codec.max_decompressed_size,
                                   codec.total_decomp_size,
                                   stream);
        break;
      case parquet::BROTLI:
        gpu_debrotli(d_comp_in,
                     d_comp_out,
                     d_comp_res_view,
                     debrotli_scratch.data(),
                     debrotli_scratch.size(),
                     stream);
        break;
      default: CUDF_FAIL("Unexpected decompression dispatch"); break;
    }
    start_pos += codec.num_pages;
  }

  CUDF_EXPECTS(thrust::all_of(rmm::exec_policy(stream),
                              comp_res.begin(),
                              comp_res.end(),
                              [] __device__(auto const& res) {
                                return res.status == compression_status::SUCCESS;
                              }),
               "Error during decompression");

  // now copy the uncompressed V2 def and rep level data
  if (not copy_in.empty()) {
    auto const d_copy_in  = cudf::detail::make_device_uvector_async(copy_in, stream);
    auto const d_copy_out = cudf::detail::make_device_uvector_async(copy_out, stream);

    gpu_copy_uncompressed_blocks(d_copy_in, d_copy_out, stream);
    stream.synchronize();
  }

  // Update the page information in device memory with the updated value of
  // page_data; it now points to the uncompressed data buffer
  pages.host_to_device(stream);

  return decomp_pages;
}

}  // namespace

void reader::impl::allocate_nesting_info()
{
  auto const& chunks      = _file_itm_data.chunks;
  auto& pages             = _file_itm_data.pages_info;
  auto& page_nesting_info = _file_itm_data.page_nesting_info;

  // compute total # of page_nesting infos needed and allocate space. doing this in one
  // buffer to keep it to a single gpu allocation
  size_t const total_page_nesting_infos = std::accumulate(
    chunks.host_ptr(), chunks.host_ptr() + chunks.size(), 0, [&](int total, auto& chunk) {
      // the schema of the input column
      auto const& schema                    = _metadata->get_schema(chunk.src_col_schema);
      auto const per_page_nesting_info_size = max(
        schema.max_definition_level + 1, _metadata->get_output_nesting_depth(chunk.src_col_schema));
      return total + (per_page_nesting_info_size * chunk.num_data_pages);
    });

  page_nesting_info = hostdevice_vector<gpu::PageNestingInfo>{total_page_nesting_infos, _stream};

  // retrieve from the gpu so we can update
  pages.device_to_host(_stream, true);

  // update pointers in the PageInfos
  int target_page_index = 0;
  int src_info_index    = 0;
  for (size_t idx = 0; idx < chunks.size(); idx++) {
    int src_col_schema                    = chunks[idx].src_col_schema;
    auto& schema                          = _metadata->get_schema(src_col_schema);
    auto const per_page_nesting_info_size = std::max(
      schema.max_definition_level + 1, _metadata->get_output_nesting_depth(src_col_schema));

    // skip my dict pages
    target_page_index += chunks[idx].num_dict_pages;
    for (int p_idx = 0; p_idx < chunks[idx].num_data_pages; p_idx++) {
      pages[target_page_index + p_idx].nesting = page_nesting_info.device_ptr() + src_info_index;
      pages[target_page_index + p_idx].num_nesting_levels = per_page_nesting_info_size;

      src_info_index += per_page_nesting_info_size;
    }
    target_page_index += chunks[idx].num_data_pages;
  }

  // copy back to the gpu
  pages.host_to_device(_stream);

  // fill in
  int nesting_info_index = 0;
  std::map<int, std::pair<std::vector<int>, std::vector<int>>> depth_remapping;
  for (size_t idx = 0; idx < chunks.size(); idx++) {
    int src_col_schema = chunks[idx].src_col_schema;

    // schema of the input column
    auto& schema = _metadata->get_schema(src_col_schema);
    // real depth of the output cudf column hierarchy (1 == no nesting, 2 == 1 level, etc)
    int max_depth = _metadata->get_output_nesting_depth(src_col_schema);

    // # of nesting infos stored per page for this column
    auto const per_page_nesting_info_size = std::max(schema.max_definition_level + 1, max_depth);

    // if this column has lists, generate depth remapping
    std::map<int, std::pair<std::vector<int>, std::vector<int>>> depth_remapping;
    if (schema.max_repetition_level > 0) {
      generate_depth_remappings(depth_remapping, src_col_schema, *_metadata);
    }

    // fill in host-side nesting info
    int schema_idx  = src_col_schema;
    auto cur_schema = _metadata->get_schema(schema_idx);
    int cur_depth   = max_depth - 1;
    while (schema_idx > 0) {
      // stub columns (basically the inner field of a list scheme element) are not real columns.
      // we can ignore them for the purposes of output nesting info
      if (!cur_schema.is_stub()) {
        // initialize each page within the chunk
        for (int p_idx = 0; p_idx < chunks[idx].num_data_pages; p_idx++) {
          gpu::PageNestingInfo* pni =
            &page_nesting_info[nesting_info_index + (p_idx * per_page_nesting_info_size)];

          // if we have lists, set our start and end depth remappings
          if (schema.max_repetition_level > 0) {
            auto remap = depth_remapping.find(src_col_schema);
            CUDF_EXPECTS(remap != depth_remapping.end(),
                         "Could not find depth remapping for schema");
            std::vector<int> const& rep_depth_remap = (remap->second.first);
            std::vector<int> const& def_depth_remap = (remap->second.second);

            for (size_t m = 0; m < rep_depth_remap.size(); m++) {
              pni[m].start_depth = rep_depth_remap[m];
            }
            for (size_t m = 0; m < def_depth_remap.size(); m++) {
              pni[m].end_depth = def_depth_remap[m];
            }
          }

          // values indexed by output column index
          pni[cur_depth].max_def_level = cur_schema.max_definition_level;
          pni[cur_depth].max_rep_level = cur_schema.max_repetition_level;
          pni[cur_depth].size          = 0;
          pni[cur_depth].type =
            to_type_id(cur_schema, _strings_to_categorical, _timestamp_type.id());
          pni[cur_depth].nullable = cur_schema.repetition_type == OPTIONAL;
        }

        // move up the hierarchy
        cur_depth--;
      }

      // next schema
      schema_idx = cur_schema.parent_idx;
      cur_schema = _metadata->get_schema(schema_idx);
    }

    nesting_info_index += (per_page_nesting_info_size * chunks[idx].num_data_pages);
  }

  // copy nesting info to the device
  page_nesting_info.host_to_device(_stream);
}

void reader::impl::load_and_decompress_data(std::vector<row_group_info> const& row_groups_info,
                                            size_type num_rows)
{
  // This function should never be called if `num_rows == 0`.
  CUDF_EXPECTS(num_rows > 0, "Number of reading rows must not be zero.");

  auto& raw_page_data    = _file_itm_data.raw_page_data;
  auto& decomp_page_data = _file_itm_data.decomp_page_data;
  auto& chunks           = _file_itm_data.chunks;
  auto& pages_info       = _file_itm_data.pages_info;

  // Descriptors for all the chunks that make up the selected columns
  const auto num_input_columns = _input_columns.size();
  const auto num_chunks        = row_groups_info.size() * num_input_columns;
  chunks                       = hostdevice_vector<gpu::ColumnChunkDesc>(0, num_chunks, _stream);

  // Association between each column chunk and its source
  std::vector<size_type> chunk_source_map(num_chunks);

  // Tracker for eventually deallocating compressed and uncompressed data
  raw_page_data = std::vector<std::unique_ptr<datasource::buffer>>(num_chunks);

  // Keep track of column chunk file offsets
  std::vector<size_t> column_chunk_offsets(num_chunks);

  // Initialize column chunk information
  size_t total_decompressed_size = 0;
  auto remaining_rows            = num_rows;
  std::vector<std::future<void>> read_rowgroup_tasks;
  for (const auto& rg : row_groups_info) {
    const auto& row_group       = _metadata->get_row_group(rg.index, rg.source_index);
    auto const row_group_start  = rg.start_row;
    auto const row_group_source = rg.source_index;
    auto const row_group_rows   = std::min<int>(remaining_rows, row_group.num_rows);
    auto const io_chunk_idx     = chunks.size();

    // generate ColumnChunkDesc objects for everything to be decoded (all input columns)
    for (size_t i = 0; i < num_input_columns; ++i) {
      auto col = _input_columns[i];
      // look up metadata
      auto& col_meta = _metadata->get_column_metadata(rg.index, rg.source_index, col.schema_idx);
      auto& schema   = _metadata->get_schema(col.schema_idx);

      auto [type_width, clock_rate, converted_type] =
        conversion_info(to_type_id(schema, _strings_to_categorical, _timestamp_type.id()),
                        _timestamp_type.id(),
                        schema.type,
                        schema.converted_type,
                        schema.type_length);

      column_chunk_offsets[chunks.size()] =
        (col_meta.dictionary_page_offset != 0)
          ? std::min(col_meta.data_page_offset, col_meta.dictionary_page_offset)
          : col_meta.data_page_offset;

      chunks.push_back(gpu::ColumnChunkDesc(col_meta.total_compressed_size,
                                            nullptr,
                                            col_meta.num_values,
                                            schema.type,
                                            type_width,
                                            row_group_start,
                                            row_group_rows,
                                            schema.max_definition_level,
                                            schema.max_repetition_level,
                                            _metadata->get_output_nesting_depth(col.schema_idx),
                                            required_bits(schema.max_definition_level),
                                            required_bits(schema.max_repetition_level),
                                            col_meta.codec,
                                            converted_type,
                                            schema.logical_type,
                                            schema.decimal_precision,
                                            clock_rate,
                                            i,
                                            col.schema_idx));

      // Map each column chunk to its column index and its source index
      chunk_source_map[chunks.size() - 1] = row_group_source;

      if (col_meta.codec != Compression::UNCOMPRESSED) {
        total_decompressed_size += col_meta.total_uncompressed_size;
      }
    }
    // Read compressed chunk data to device memory
    read_rowgroup_tasks.push_back(read_column_chunks_async(_sources,
                                                           raw_page_data,
                                                           chunks,
                                                           io_chunk_idx,
                                                           chunks.size(),
                                                           column_chunk_offsets,
                                                           chunk_source_map,
                                                           _stream));

    remaining_rows -= row_group.num_rows;
  }
  for (auto& task : read_rowgroup_tasks) {
    task.wait();
  }

  CUDF_EXPECTS(remaining_rows <= 0, "All rows data must be read.");

  // Process dataset chunk pages into output columns
  auto const total_pages = count_page_headers(chunks, _stream);
  pages_info             = hostdevice_vector<gpu::PageInfo>(total_pages, total_pages, _stream);

  if (total_pages > 0) {
    // decoding of column/page information
    decode_page_headers(chunks, pages_info, _stream);
    if (total_decompressed_size > 0) {
      decomp_page_data = decompress_page_data(chunks, pages_info, _stream);
      // Free compressed data
      for (size_t c = 0; c < chunks.size(); c++) {
        if (chunks[c].codec != parquet::Compression::UNCOMPRESSED) {
          raw_page_data[c].reset();
          // TODO: Check if this is called
        }
      }
    }

    // build output column info
    // walk the schema, building out_buffers that mirror what our final cudf columns will look
    // like. important : there is not necessarily a 1:1 mapping between input columns and output
    // columns. For example, parquet does not explicitly store a ColumnChunkDesc for struct
    // columns. The "structiness" is simply implied by the schema.  For example, this schema:
    //  required group field_id=1 name {
    //    required binary field_id=2 firstname (String);
    //    required binary field_id=3 middlename (String);
    //    required binary field_id=4 lastname (String);
    // }
    // will only contain 3 columns of data (firstname, middlename, lastname).  But of course
    // "name" is a struct column that we want to return, so we have to make sure that we
    // create it ourselves.
    // std::vector<output_column_info> output_info = build_output_column_info();

    // nesting information (sizes, etc) stored -per page-
    // note : even for flat schemas, we allocate 1 level of "nesting" info
    allocate_nesting_info();
  }
}

namespace {

struct cumulative_row_info {
  size_t row_count;   // cumulative row count
  size_t size_bytes;  // cumulative size in bytes
  int key;            // schema index
};

#if defined(PREPROCESS_DEBUG)
void print_pages(hostdevice_vector<gpu::PageInfo>& pages, rmm::cuda_stream_view _stream)
{
  pages.device_to_host(_stream, true);
  for (size_t idx = 0; idx < pages.size(); idx++) {
    auto const& p = pages[idx];
    // skip dictionary pages
    if (p.flags & gpu::PAGEINFO_FLAGS_DICTIONARY) { continue; }
    printf(
      "P(%lu, s:%d): chunk_row(%d), num_rows(%d), skipped_values(%d), skipped_leaf_values(%d)\n",
      idx,
      p.src_col_schema,
      p.chunk_row,
      p.num_rows,
      p.skipped_values,
      p.skipped_leaf_values);
  }
}

void print_cumulative_page_info(hostdevice_vector<gpu::PageInfo>& pages,
                                rmm::device_uvector<int32_t> const& page_index,
                                rmm::device_uvector<cumulative_row_info> const& c_info,
                                rmm::cuda_stream_view stream)
{
  pages.device_to_host(stream, true);

  printf("------------\nCumulative sizes by page\n");

  std::vector<int> schemas(pages.size());
  std::vector<int> h_page_index(pages.size());
  hipMemcpy(
    h_page_index.data(), page_index.data(), sizeof(int) * pages.size(), hipMemcpyDeviceToHost);
  std::vector<cumulative_row_info> h_cinfo(pages.size());
  hipMemcpy(h_cinfo.data(),
             c_info.data(),
             sizeof(cumulative_row_info) * pages.size(),
             hipMemcpyDeviceToHost);
  auto schema_iter = cudf::detail::make_counting_transform_iterator(
    0, [&](size_type i) { return pages[h_page_index[i]].src_col_schema; });
  thrust::copy(thrust::seq, schema_iter, schema_iter + pages.size(), schemas.begin());
  auto last = thrust::unique(thrust::seq, schemas.begin(), schemas.end());
  schemas.resize(last - schemas.begin());
  printf("Num schemas: %lu\n", schemas.size());

  for (size_t idx = 0; idx < schemas.size(); idx++) {
    printf("Schema %d\n", schemas[idx]);
    for (size_t pidx = 0; pidx < pages.size(); pidx++) {
      auto const& page = pages[h_page_index[pidx]];
      if (page.flags & gpu::PAGEINFO_FLAGS_DICTIONARY || page.src_col_schema != schemas[idx]) {
        continue;
      }
      printf("\tP: {%lu, %lu}\n", h_cinfo[pidx].row_count, h_cinfo[pidx].size_bytes);
    }
  }
}

void print_cumulative_row_info(
  host_span<cumulative_row_info const> sizes,
  std::string const& label,
  std::optional<std::vector<gpu::chunk_read_info>> splits = std::nullopt)
{
  if (splits.has_value()) {
    printf("------------\nSplits\n");
    for (size_t idx = 0; idx < splits->size(); idx++) {
      printf("{%lu, %lu}\n", splits.value()[idx].skip_rows, splits.value()[idx].num_rows);
    }
  }

  printf("------------\nCumulative sizes %s\n", label.c_str());
  for (size_t idx = 0; idx < sizes.size(); idx++) {
    printf("{%lu, %lu, %d}", sizes[idx].row_count, sizes[idx].size_bytes, sizes[idx].key);
    if (splits.has_value()) {
      // if we have a split at this row count and this is the last instance of this row count
      auto start = thrust::make_transform_iterator(
        splits->begin(), [](gpu::chunk_read_info const& i) { return i.skip_rows; });
      auto end               = start + splits->size();
      auto split             = std::find(start, end, sizes[idx].row_count);
      auto const split_index = [&]() -> int {
        if (split != end &&
            ((idx == sizes.size() - 1) || (sizes[idx + 1].row_count > sizes[idx].row_count))) {
          return static_cast<int>(std::distance(start, split));
        }
        return idx == 0 ? 0 : -1;
      }();
      if (split_index >= 0) {
        printf(" <-- split {%lu, %lu}",
               splits.value()[split_index].skip_rows,
               splits.value()[split_index].num_rows);
      }
    }
    printf("\n");
  }
}
#endif  // PREPROCESS_DEBUG

/**
 * @brief Functor which reduces two cumulative_row_info structs of the same key.
 */
struct cumulative_row_sum {
  cumulative_row_info operator()
    __device__(cumulative_row_info const& a, cumulative_row_info const& b) const
  {
    return cumulative_row_info{a.row_count + b.row_count, a.size_bytes + b.size_bytes, a.key};
  }
};

/**
 * @brief Functor which computes the total data size for a given type of cudf column.
 *
 * In the case of strings, the return size does not include the chars themselves. That
 * information is tracked separately (see PageInfo::str_bytes).
 */
struct row_size_functor {
  __device__ size_t validity_size(size_t num_rows, bool nullable)
  {
    return nullable ? (cudf::util::div_rounding_up_safe(num_rows, size_t{32}) * 4) : 0;
  }

  template <typename T>
  __device__ size_t operator()(size_t num_rows, bool nullable)
  {
    auto const element_size = sizeof(device_storage_type_t<T>);
    return (element_size * num_rows) + validity_size(num_rows, nullable);
  }
};

template <>
__device__ size_t row_size_functor::operator()<list_view>(size_t num_rows, bool nullable)
{
  auto const offset_size = sizeof(offset_type);
  // NOTE: Adding the + 1 offset here isn't strictly correct.  There will only be 1 extra offset
  // for the entire column, whereas this is adding an extra offset per page.  So we will get a
  // small over-estimate of the real size of the order :  # of pages * 4 bytes. It seems better
  // to overestimate size somewhat than to underestimate it and potentially generate chunks
  // that are too large.
  return (offset_size * (num_rows + 1)) + validity_size(num_rows, nullable);
}

template <>
__device__ size_t row_size_functor::operator()<struct_view>(size_t num_rows, bool nullable)
{
  return validity_size(num_rows, nullable);
}

template <>
__device__ size_t row_size_functor::operator()<string_view>(size_t num_rows, bool nullable)
{
  // only returns the size of offsets and validity. the size of the actual string chars
  // is tracked separately.
  auto const offset_size = sizeof(offset_type);
  // see note about offsets in the list_view template.
  return (offset_size * (num_rows + 1)) + validity_size(num_rows, nullable);
}

/**
 * @brief Functor which computes the total output cudf data size for all of
 * the data in this page.
 *
 * Sums across all nesting levels.
 */
struct get_cumulative_row_info {
  gpu::PageInfo const* const pages;

  __device__ cumulative_row_info operator()(size_type index)
  {
    auto const& page = pages[index];
    if (page.flags & gpu::PAGEINFO_FLAGS_DICTIONARY) {
      return cumulative_row_info{0, 0, page.src_col_schema};
    }

    // total nested size, not counting string data
    auto iter =
      cudf::detail::make_counting_transform_iterator(0, [page, index] __device__(size_type i) {
        auto const& pni = page.nesting[i];
        return cudf::type_dispatcher(
          data_type{pni.type}, row_size_functor{}, pni.size, pni.nullable);
      });

    size_t const row_count = static_cast<size_t>(page.nesting[0].size);
    return {row_count,
            thrust::reduce(thrust::seq, iter, iter + page.num_nesting_levels) + page.str_bytes,
            page.src_col_schema};
  }
};

/**
 * @brief Functor which computes the effective size of all input columns by page.
 *
 * For a given row, we want to find the cost of all pages for all columns involved
 * in loading up to that row.  The complication here is that not all pages are the
 * same size between columns. Example:
 *
 *              page row counts
 * Column A:    0 <----> 100 <----> 200
 * Column B:    0 <---------------> 200 <--------> 400
                          |
 * if we decide to split at row 100, we don't really know the actual amount of bytes in column B
 * at that point.  So we have to proceed as if we are taking the bytes from all 200 rows of that
 * page. Essentially, a conservative over-estimate of the real size.
 */
struct row_total_size {
  cumulative_row_info const* c_info;
  size_type const* key_offsets;
  size_t num_keys;

  __device__ cumulative_row_info operator()(cumulative_row_info const& i)
  {
    // sum sizes for each input column at this row
    size_t sum = 0;
    for (int idx = 0; idx < num_keys; idx++) {
      auto const start = key_offsets[idx];
      auto const end   = key_offsets[idx + 1];
      auto iter        = cudf::detail::make_counting_transform_iterator(
        0, [&] __device__(size_type i) { return c_info[i].row_count; });
      auto const page_index =
        thrust::lower_bound(thrust::seq, iter + start, iter + end, i.row_count) - iter;
      sum += c_info[page_index].size_bytes;
    }
    return {i.row_count, sum, i.key};
  }
};

/**
 * @brief Given a vector of cumulative {row_count, byte_size} pairs and a chunk read
 * limit, determine the set of splits.
 *
 * @param sizes Vector of cumulative {row_count, byte_size} pairs
 * @param num_rows Total number of rows to read
 * @param chunk_read_limit Limit on total number of bytes to be returned per read, for all columns
 */
std::vector<gpu::chunk_read_info> find_splits(std::vector<cumulative_row_info> const& sizes,
                                              size_t num_rows,
                                              size_t chunk_read_limit)
{
  // now we have an array of {row_count, real output bytes}. just walk through it and generate
  // splits.
  // TODO: come up with a clever way to do this entirely in parallel. For now, as long as batch
  // sizes are reasonably large, this shouldn't iterate too many times
  std::vector<gpu::chunk_read_info> splits;
  {
    size_t cur_pos             = 0;
    size_t cur_cumulative_size = 0;
    size_t cur_row_count       = 0;
    auto start = thrust::make_transform_iterator(sizes.begin(), [&](cumulative_row_info const& i) {
      return i.size_bytes - cur_cumulative_size;
    });
    auto end   = start + sizes.size();
    while (cur_row_count < num_rows) {
      int64_t split_pos =
        thrust::lower_bound(thrust::seq, start + cur_pos, end, chunk_read_limit) - start;

      // if we're past the end, or if the returned bucket is > than the chunk_read_limit, move back
      // one.
      if (static_cast<size_t>(split_pos) >= sizes.size() ||
          (sizes[split_pos].size_bytes - cur_cumulative_size > chunk_read_limit)) {
        split_pos--;
      }

      // best-try. if we can't find something that'll fit, we have to go bigger. we're doing this in
      // a loop because all of the cumulative sizes for all the pages are sorted into one big list.
      // so if we had two columns, both of which had an entry {1000, 10000}, that entry would be in
      // the list twice. so we have to iterate until we skip past all of them.  The idea is that we
      // either do this, or we have to call unique() on the input first.
      while (split_pos < (static_cast<int64_t>(sizes.size()) - 1) &&
             (split_pos < 0 || sizes[split_pos].row_count == cur_row_count)) {
        split_pos++;
      }

      auto const start_row = cur_row_count;
      cur_row_count        = sizes[split_pos].row_count;
      splits.push_back(gpu::chunk_read_info{start_row, cur_row_count - start_row});
      cur_pos             = split_pos;
      cur_cumulative_size = sizes[split_pos].size_bytes;
    }
  }
  // print_cumulative_row_info(sizes, "adjusted", splits);

  return splits;
}

/**
 * @brief Given a set of pages that have had their sizes computed by nesting level and
 * a limit on total read size, generate a set of {skip_rows, num_rows} pairs representing
 * a set of reads that will generate output columns of total size <= `chunk_read_limit` bytes.
 *
 * @param pages All pages in the file
 * @param id Additional intermediate information required to process the pages
 * @param num_rows Total number of rows to read
 * @param chunk_read_limit Limit on total number of bytes to be returned per read, for all columns
 * @param stream CUDA stream to use, default 0
 */
std::vector<gpu::chunk_read_info> compute_splits(hostdevice_vector<gpu::PageInfo>& pages,
                                                 gpu::chunk_intermediate_data const& id,
                                                 size_t num_rows,
                                                 size_t chunk_read_limit,
                                                 rmm::cuda_stream_view stream)
{
  auto const& page_keys  = id.page_keys;
  auto const& page_index = id.page_index;

  // generate cumulative row counts and sizes
  rmm::device_uvector<cumulative_row_info> c_info(page_keys.size(), stream);
  // convert PageInfo to cumulative_row_info
  auto page_input = thrust::make_transform_iterator(page_index.begin(),
                                                    get_cumulative_row_info{pages.device_ptr()});
  thrust::inclusive_scan_by_key(rmm::exec_policy(stream),
                                page_keys.begin(),
                                page_keys.end(),
                                page_input,
                                c_info.begin(),
                                thrust::equal_to{},
                                cumulative_row_sum{});
  // print_cumulative_page_info(pages, page_index, c_info, stream);

  // sort by row count
  rmm::device_uvector<cumulative_row_info> c_info_sorted{c_info, stream};
  thrust::sort(rmm::exec_policy(stream),
               c_info_sorted.begin(),
               c_info_sorted.end(),
               [] __device__(cumulative_row_info const& a, cumulative_row_info const& b) {
                 return a.row_count < b.row_count;
               });

  std::vector<cumulative_row_info> h_c_info_sorted(c_info_sorted.size());
  hipMemcpy(h_c_info_sorted.data(),
             c_info_sorted.data(),
             sizeof(cumulative_row_info) * c_info_sorted.size(),
             hipMemcpyDeviceToHost);
  // print_cumulative_row_info(h_c_info_sorted, "raw");

  // generate key offsets (offsets to the start of each partition of keys). worst case is 1 page per
  // key
  rmm::device_uvector<size_type> key_offsets(page_keys.size() + 1, stream);
  auto const key_offsets_end = thrust::reduce_by_key(rmm::exec_policy(stream),
                                                     page_keys.begin(),
                                                     page_keys.end(),
                                                     thrust::make_constant_iterator(1),
                                                     thrust::make_discard_iterator(),
                                                     key_offsets.begin())
                                 .second;
  size_t const num_unique_keys = key_offsets_end - key_offsets.begin();
  thrust::exclusive_scan(
    rmm::exec_policy(stream), key_offsets.begin(), key_offsets.end(), key_offsets.begin());

  // adjust the cumulative info such that for each row count, the size includes any pages that span
  // that row count. this is so that if we have this case:
  //              page row counts
  // Column A:    0 <----> 100 <----> 200
  // Column B:    0 <---------------> 200 <--------> 400
  //                        |
  // if we decide to split at row 100, we don't really know the actual amount of bytes in column B
  // at that point.  So we have to proceed as if we are taking the bytes from all 200 rows of that
  // page.
  //
  rmm::device_uvector<cumulative_row_info> aggregated_info(c_info.size(), stream);
  thrust::transform(rmm::exec_policy(stream),
                    c_info_sorted.begin(),
                    c_info_sorted.end(),
                    aggregated_info.begin(),
                    row_total_size{c_info.data(), key_offsets.data(), num_unique_keys});

  // bring back to the cpu
  std::vector<cumulative_row_info> h_aggregated_info(aggregated_info.size());
  hipMemcpyAsync(h_aggregated_info.data(),
                  aggregated_info.data(),
                  sizeof(cumulative_row_info) * c_info.size(),
                  hipMemcpyDeviceToHost,
                  stream);
  stream.synchronize();

  return find_splits(h_aggregated_info, num_rows, chunk_read_limit);
}

struct get_page_chunk_idx {
  __device__ size_type operator()(gpu::PageInfo const& page) { return page.chunk_idx; }
};

struct get_page_num_rows {
  __device__ size_type operator()(gpu::PageInfo const& page) { return page.num_rows; }
};

struct get_page_schema {
  __device__ size_type operator()(gpu::PageInfo const& page) { return page.src_col_schema; }
};

/**
 * @brief Returns the size field of a PageInfo struct for a given depth, keyed by schema.
 */
struct get_page_nesting_size {
  size_type const src_col_schema;
  size_type const depth;
  gpu::PageInfo const* const pages;

  __device__ size_type operator()(int index) const
  {
    auto const& page = pages[index];
    if (page.src_col_schema != src_col_schema || page.flags & gpu::PAGEINFO_FLAGS_DICTIONARY) {
      return 0;
    }
    return page.nesting[depth].batch_size;
  }
};

/**
 * @brief Writes to the chunk_row field of the PageInfo struct.
 */
struct chunk_row_output_iter {
  gpu::PageInfo* p;
  using value_type        = size_type;
  using difference_type   = size_type;
  using pointer           = size_type*;
  using reference         = size_type&;
  using iterator_category = thrust::output_device_iterator_tag;

  __host__ __device__ chunk_row_output_iter operator+(int i)
  {
    return chunk_row_output_iter{p + i};
  }

  __host__ __device__ void operator++() { p++; }

  __device__ reference operator[](int i) { return p[i].chunk_row; }
  __device__ reference operator*() { return p->chunk_row; }
};

/**
 * @brief Writes to the page_start_value field of the PageNestingInfo struct, keyed by schema.
 */
struct start_offset_output_iterator {
  gpu::PageInfo* pages;
  int const* page_indices;
  int cur_index;
  int src_col_schema;
  int nesting_depth;
  int empty               = 0;
  using value_type        = size_type;
  using difference_type   = size_type;
  using pointer           = size_type*;
  using reference         = size_type&;
  using iterator_category = thrust::output_device_iterator_tag;

  constexpr void operator=(start_offset_output_iterator const& other)
  {
    pages          = other.pages;
    page_indices   = other.page_indices;
    cur_index      = other.cur_index;
    src_col_schema = other.src_col_schema;
    nesting_depth  = other.nesting_depth;
  }

  constexpr start_offset_output_iterator operator+(int i)
  {
    return start_offset_output_iterator{
      pages, page_indices, cur_index + i, src_col_schema, nesting_depth};
  }

  constexpr void operator++() { cur_index++; }

  __device__ reference operator[](int i) { return dereference(cur_index + i); }
  __device__ reference operator*() { return dereference(cur_index); }

 private:
  __device__ reference dereference(int index)
  {
    gpu::PageInfo const& p = pages[page_indices[index]];
    if (p.src_col_schema != src_col_schema || p.flags & gpu::PAGEINFO_FLAGS_DICTIONARY) {
      return empty;
    }
    return p.nesting[nesting_depth].page_start_value;
  }
};

struct flat_column_num_rows {
  gpu::PageInfo const* pages;
  gpu::ColumnChunkDesc const* chunks;

  __device__ size_type operator()(size_type pindex) const
  {
    gpu::PageInfo const& page = pages[pindex];
    // ignore dictionary pages and pages belonging to any column containing repetition (lists)
    if ((page.flags & gpu::PAGEINFO_FLAGS_DICTIONARY) ||
        (chunks[page.chunk_idx].max_level[gpu::level_type::REPETITION] > 0)) {
      return 0;
    }
    return page.num_rows;
  }
};

struct row_counts_nonzero {
  __device__ bool operator()(size_type count) const { return count > 0; }
};

struct row_counts_different {
  size_type const expected;
  __device__ bool operator()(size_type count) const { return (count != 0) && (count != expected); }
};

/**
 * @brief Detect malformed parquet input data.
 *
 * We have seen cases where parquet files can be oddly malformed. This function specifically
 * detects one case in particular:
 *
 * - When you have a file containing N rows
 * - For some reason, the sum total of the number of rows over all pages for a given column
 *   is != N
 *
 * @param pages All pages to be decoded
 * @param chunks Chunk data
 * @param page_keys Keys (schema id) associated with each page, sorted by column
 * @param page_index Page indices for iteration, sorted by column
 * @param expected_row_count Expected row count, if applicable
 * @param stream CUDA stream used for device memory operations and kernel launches
 */
void detect_malformed_pages(hostdevice_vector<gpu::PageInfo>& pages,
                            hostdevice_vector<gpu::ColumnChunkDesc> const& chunks,
                            device_span<const int> page_keys,
                            device_span<const int> page_index,
                            std::optional<size_t> expected_row_count,
                            rmm::cuda_stream_view stream)
{
  // sum row counts for all non-dictionary, non-list columns. other columns will be indicated as 0
  rmm::device_uvector<size_type> row_counts(pages.size(),
                                            stream);  // worst case:  num keys == num pages
  auto const size_iter = thrust::make_transform_iterator(
    page_index.begin(), flat_column_num_rows{pages.device_ptr(), chunks.device_ptr()});
  auto const row_counts_begin = row_counts.begin();
  auto const row_counts_end   = thrust::reduce_by_key(rmm::exec_policy(stream),
                                                    page_keys.begin(),
                                                    page_keys.end(),
                                                    size_iter,
                                                    thrust::make_discard_iterator(),
                                                    row_counts_begin)
                                .second;

  // make sure all non-zero row counts are the same
  rmm::device_uvector<size_type> compacted_row_counts(pages.size(), stream);
  auto const compacted_row_counts_begin = compacted_row_counts.begin();
  auto const compacted_row_counts_end   = thrust::copy_if(rmm::exec_policy(stream),
                                                        row_counts_begin,
                                                        row_counts_end,
                                                        compacted_row_counts_begin,
                                                        row_counts_nonzero{});
  if (compacted_row_counts_end != compacted_row_counts_begin) {
    size_t const found_row_count = static_cast<size_t>(compacted_row_counts.element(0, stream));

    // if we somehow don't match the expected row count from the row groups themselves
    if (expected_row_count.has_value()) {
      CUDF_EXPECTS(expected_row_count.value() == found_row_count,
                   "Encountered malformed parquet page data (unexpected row count in page data)");
    }

    // all non-zero row counts must be the same
    auto const chk =
      thrust::count_if(rmm::exec_policy(stream),
                       compacted_row_counts_begin,
                       compacted_row_counts_end,
                       row_counts_different{static_cast<size_type>(found_row_count)});
    CUDF_EXPECTS(chk == 0,
                 "Encountered malformed parquet page data (row count mismatch in page data)");
  }
}

}  // anonymous namespace

void reader::impl::preprocess_pages(size_t skip_rows,
                                    size_t num_rows,
                                    bool uses_custom_row_bounds,
                                    size_t chunk_read_limit)
{
  auto& chunks = _file_itm_data.chunks;
  auto& pages  = _file_itm_data.pages_info;

  // compute page ordering.
  //
  // ordering of pages is by input column schema, repeated across row groups.  so
  // if we had 3 columns, each with 2 pages, and 1 row group, our schema values might look like
  //
  // 1, 1, 2, 2, 3, 3
  //
  // However, if we had more than one row group, the pattern would be
  //
  // 1, 1, 2, 2, 3, 3, 1, 1, 2, 2, 3, 3
  // ^ row group 0     |
  //                   ^ row group 1
  //
  // To process pages by key (exclusive_scan_by_key, reduce_by_key, etc), the ordering we actually
  // want is
  //
  // 1, 1, 1, 1, 2, 2, 2, 2, 3, 3, 3, 3
  //
  // We also need to preserve key-relative page ordering, so we need to use a stable sort.
  rmm::device_uvector<int> page_keys(pages.size(), _stream);
  rmm::device_uvector<int> page_index(pages.size(), _stream);
  {
    thrust::transform(rmm::exec_policy(_stream),
                      pages.device_ptr(),
                      pages.device_ptr() + pages.size(),
                      page_keys.begin(),
                      get_page_schema{});

    thrust::sequence(rmm::exec_policy(_stream), page_index.begin(), page_index.end());
    thrust::stable_sort_by_key(rmm::exec_policy(_stream),
                               page_keys.begin(),
                               page_keys.end(),
                               page_index.begin(),
                               thrust::less<int>());
  }

  // detect malformed columns.
  // - we have seen some cases in the wild where we have a row group containing N
  //   rows, but the total number of rows in the pages for column X is != N. while it
  //   is possible to load this by just capping the number of rows read, we cannot tell
  //   which rows are invalid so we may be returning bad data. in addition, this mismatch
  //   confuses the chunked reader
  detect_malformed_pages(pages,
                         chunks,
                         page_keys,
                         page_index,
                         uses_custom_row_bounds ? std::nullopt : std::make_optional(num_rows),
                         _stream);

  // iterate over all input columns and determine if they contain lists so we can further
  // preprocess them.
  bool has_lists = false;
  for (size_t idx = 0; idx < _input_columns.size(); idx++) {
    auto const& input_col  = _input_columns[idx];
    size_t const max_depth = input_col.nesting_depth();

    auto* cols = &_output_buffers;
    for (size_t l_idx = 0; l_idx < max_depth; l_idx++) {
      auto& out_buf = (*cols)[input_col.nesting[l_idx]];
      cols          = &out_buf.children;

      // if this has a list parent, we have to get column sizes from the
      // data computed during gpu::ComputePageSizes
      if (out_buf.user_data & PARQUET_COLUMN_BUFFER_FLAG_HAS_LIST_PARENT) {
        has_lists = true;
        break;
      }
    }
    if (has_lists) { break; }
  }

  // generate string dict indices if necessary
  {
    auto is_dict_chunk = [](const gpu::ColumnChunkDesc& chunk) {
      return (chunk.data_type & 0x7) == BYTE_ARRAY && chunk.num_dict_pages > 0;
    };

    // Count the number of string dictionary entries
    // NOTE: Assumes first page in the chunk is always the dictionary page
    size_t total_str_dict_indexes = 0;
    for (size_t c = 0, page_count = 0; c < chunks.size(); c++) {
      if (is_dict_chunk(chunks[c])) {
        total_str_dict_indexes += pages[page_count].num_input_values;
      }
      page_count += chunks[c].max_num_pages;
    }

    // Build index for string dictionaries since they can't be indexed
    // directly due to variable-sized elements
    _chunk_itm_data.str_dict_index =
      cudf::detail::make_zeroed_device_uvector_async<string_index_pair>(total_str_dict_indexes,
                                                                        _stream);

    // Update chunks with pointers to string dict indices
    for (size_t c = 0, page_count = 0, str_ofs = 0; c < chunks.size(); c++) {
      input_column_info const& input_col = _input_columns[chunks[c].src_col_index];
      CUDF_EXPECTS(input_col.schema_idx == chunks[c].src_col_schema,
                   "Column/page schema index mismatch");
      if (is_dict_chunk(chunks[c])) {
        chunks[c].str_dict_index = _chunk_itm_data.str_dict_index.data() + str_ofs;
        str_ofs += pages[page_count].num_input_values;
      }

      // column_data_base will always point to leaf data, even for nested types.
      page_count += chunks[c].max_num_pages;
    }

    if (total_str_dict_indexes > 0) {
      chunks.host_to_device(_stream);
      gpu::BuildStringDictionaryIndex(chunks.device_ptr(), chunks.size(), _stream);
    }
  }

  // intermediate data we will need for further chunked reads
  if (has_lists || chunk_read_limit > 0) {
    // computes:
    // PageNestingInfo::num_rows for each page. the true number of rows (taking repetition into
    // account), not just the number of values. PageNestingInfo::size for each level of nesting, for
    // each page.
    //
    // we will be applying a later "trim" pass if skip_rows/num_rows is being used, which can happen
    // if:
    // - user has passed custom row bounds
    // - we will be doing a chunked read
    gpu::ComputePageSizes(pages,
                          chunks,
                          0,  // 0-max size_t. process all possible rows
                          std::numeric_limits<size_t>::max(),
                          true,                  // compute num_rows
                          chunk_read_limit > 0,  // compute string sizes
                          _stream);

    // computes:
    // PageInfo::chunk_row (the absolute start row index) for all pages
    // Note: this is doing some redundant work for pages in flat hierarchies.  chunk_row has already
    // been computed during header decoding. the overall amount of work here is very small though.
    auto key_input  = thrust::make_transform_iterator(pages.device_ptr(), get_page_chunk_idx{});
    auto page_input = thrust::make_transform_iterator(pages.device_ptr(), get_page_num_rows{});
    thrust::exclusive_scan_by_key(rmm::exec_policy(_stream),
                                  key_input,
                                  key_input + pages.size(),
                                  page_input,
                                  chunk_row_output_iter{pages.device_ptr()});

    // preserve page ordering data
    _chunk_itm_data.page_keys  = std::move(page_keys);
    _chunk_itm_data.page_index = std::move(page_index);

    // retrieve pages back
    pages.device_to_host(_stream, true);

    // print_pages(pages, _stream);
  }

  // compute splits if necessary. otherwise return a single split representing
  // the whole file.
  _chunk_read_info = chunk_read_limit > 0
                       ? compute_splits(pages, _chunk_itm_data, num_rows, chunk_read_limit, _stream)
                       : std::vector<gpu::chunk_read_info>{{skip_rows, num_rows}};
}

void reader::impl::allocate_columns(size_t skip_rows, size_t num_rows, bool uses_custom_row_bounds)
{
  auto const& chunks = _file_itm_data.chunks;
  auto& pages        = _file_itm_data.pages_info;

  // Should not reach here if there is no page data.
  CUDF_EXPECTS(pages.size() > 0, "There is no page to parse");

  // computes:
  // PageNestingInfo::batch_size for each level of nesting, for each page, taking row bounds into
  // account. PageInfo::skipped_values, which tells us where to start decoding in the input to
  // respect the user bounds. It is only necessary to do this second pass if uses_custom_row_bounds
  // is set (if the user has specified artificial bounds).
  if (uses_custom_row_bounds) {
    gpu::ComputePageSizes(pages,
                          chunks,
                          skip_rows,
                          num_rows,
                          false,  // num_rows is already computed
                          false,  // no need to compute string sizes
                          _stream);

    // print_pages(pages, _stream);
  }

  // iterate over all input columns and allocate any associated output
  // buffers if they are not part of a list hierarchy. mark down
  // if we have any list columns that need further processing.
  bool has_lists = false;
  for (size_t idx = 0; idx < _input_columns.size(); idx++) {
    auto const& input_col  = _input_columns[idx];
    size_t const max_depth = input_col.nesting_depth();

    auto* cols = &_output_buffers;
    for (size_t l_idx = 0; l_idx < max_depth; l_idx++) {
      auto& out_buf = (*cols)[input_col.nesting[l_idx]];
      cols          = &out_buf.children;

      // if this has a list parent, we have to get column sizes from the
      // data computed during gpu::ComputePageSizes
      if (out_buf.user_data & PARQUET_COLUMN_BUFFER_FLAG_HAS_LIST_PARENT) {
        has_lists = true;
      }
      // if we haven't already processed this column because it is part of a struct hierarchy
      else if (out_buf.size == 0) {
        // add 1 for the offset if this is a list column
        out_buf.create(
          out_buf.type.id() == type_id::LIST && l_idx < max_depth ? num_rows + 1 : num_rows,
          _stream,
          _mr);
      }
    }
  }

  // compute output column sizes by examining the pages of the -input- columns
  if (has_lists) {
    auto& page_keys  = _chunk_itm_data.page_keys;
    auto& page_index = _chunk_itm_data.page_index;
    for (size_t idx = 0; idx < _input_columns.size(); idx++) {
      auto const& input_col = _input_columns[idx];
      auto src_col_schema   = input_col.schema_idx;
      size_t max_depth      = input_col.nesting_depth();

      auto* cols = &_output_buffers;
      for (size_t l_idx = 0; l_idx < input_col.nesting_depth(); l_idx++) {
        auto& out_buf = (*cols)[input_col.nesting[l_idx]];
        cols          = &out_buf.children;

        // size iterator. indexes pages by sorted order
        auto size_input = thrust::make_transform_iterator(
          page_index.begin(),
          get_page_nesting_size{src_col_schema, static_cast<size_type>(l_idx), pages.device_ptr()});

        // if this buffer is part of a list hierarchy, we need to determine it's
        // final size and allocate it here.
        //
        // for struct columns, higher levels of the output columns are shared between input
        // columns. so don't compute any given level more than once.
        if ((out_buf.user_data & PARQUET_COLUMN_BUFFER_FLAG_HAS_LIST_PARENT) && out_buf.size == 0) {
          int size =
            thrust::reduce(rmm::exec_policy(_stream), size_input, size_input + pages.size());

          // if this is a list column add 1 for non-leaf levels for the terminating offset
          if (out_buf.type.id() == type_id::LIST && l_idx < max_depth) { size++; }

          // allocate
          out_buf.create(size, _stream, _mr);
        }

        // for nested hierarchies, compute per-page start offset
        if (input_col.has_repetition) {
          thrust::exclusive_scan_by_key(
            rmm::exec_policy(_stream),
            page_keys.begin(),
            page_keys.end(),
            size_input,
            start_offset_output_iterator{pages.device_ptr(),
                                         page_index.begin(),
                                         0,
                                         static_cast<int>(src_col_schema),
                                         static_cast<int>(l_idx)});
        }
      }
    }
  }
}

}  // namespace cudf::io::detail::parquet
