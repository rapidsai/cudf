#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2022-2024, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "error.hpp"
#include "reader_impl.hpp"

#include <cudf/detail/iterator.cuh>
#include <cudf/detail/nvtx/ranges.hpp>
#include <cudf/detail/utilities/integer_utils.hpp>
#include <cudf/detail/utilities/vector_factories.hpp>

#include <rmm/exec_policy.hpp>

#include <cuda/functional>
#include <thrust/binary_search.h>
#include <thrust/fill.h>
#include <thrust/functional.h>
#include <thrust/iterator/constant_iterator.h>
#include <thrust/iterator/discard_iterator.h>
#include <thrust/iterator/iterator_categories.h>
#include <thrust/iterator/transform_iterator.h>
#include <thrust/logical.h>
#include <thrust/reduce.h>
#include <thrust/scan.h>
#include <thrust/sequence.h>
#include <thrust/sort.h>
#include <thrust/transform.h>
#include <thrust/transform_scan.h>
#include <thrust/unique.h>

#include <bitset>
#include <numeric>

namespace cudf::io::parquet::detail {
namespace {

#if defined(PREPROCESS_DEBUG)
void print_pages(cudf::detail::hostdevice_vector<PageInfo>& pages, rmm::cuda_stream_view _stream)
{
  pages.device_to_host_sync(_stream);
  for (size_t idx = 0; idx < pages.size(); idx++) {
    auto const& p = pages[idx];
    // skip dictionary pages
    if (p.flags & PAGEINFO_FLAGS_DICTIONARY) { continue; }
    printf(
      "P(%lu, s:%d): chunk_row(%d), num_rows(%d), skipped_values(%d), skipped_leaf_values(%d), "
      "str_bytes(%d)\n",
      idx,
      p.src_col_schema,
      p.chunk_row,
      p.num_rows,
      p.skipped_values,
      p.skipped_leaf_values,
      p.str_bytes);
  }
}
#endif  // PREPROCESS_DEBUG

/**
 * @brief Generate depth remappings for repetition and definition levels.
 *
 * When dealing with columns that contain lists, we must examine incoming
 * repetition and definition level pairs to determine what range of output nesting
 * is indicated when adding new values.  This function generates the mappings of
 * the R/D levels to those start/end bounds
 *
 * @param remap Maps column schema index to the R/D remapping vectors for that column
 * @param src_col_schema The column schema to generate the new mapping for
 * @param md File metadata information
 */
void generate_depth_remappings(std::map<int, std::pair<std::vector<int>, std::vector<int>>>& remap,
                               int src_col_schema,
                               aggregate_reader_metadata const& md)
{
  // already generated for this level
  if (remap.find(src_col_schema) != remap.end()) { return; }
  auto schema   = md.get_schema(src_col_schema);
  int max_depth = md.get_output_nesting_depth(src_col_schema);

  CUDF_EXPECTS(remap.find(src_col_schema) == remap.end(),
               "Attempting to remap a schema more than once");
  auto inserted =
    remap.insert(std::pair<int, std::pair<std::vector<int>, std::vector<int>>>{src_col_schema, {}});
  auto& depth_remap = inserted.first->second;

  std::vector<int>& rep_depth_remap = (depth_remap.first);
  rep_depth_remap.resize(schema.max_repetition_level + 1);
  std::vector<int>& def_depth_remap = (depth_remap.second);
  def_depth_remap.resize(schema.max_definition_level + 1);

  // the key:
  // for incoming level values  R/D
  // add values starting at the shallowest nesting level X has repetition level R
  // until you reach the deepest nesting level Y that corresponds to the repetition level R1
  // held by the nesting level that has definition level D
  //
  // Example: a 3 level struct with a list at the bottom
  //
  //                     R / D   Depth
  // level0              0 / 1     0
  //   level1            0 / 2     1
  //     level2          0 / 3     2
  //       list          0 / 3     3
  //         element     1 / 4     4
  //
  // incoming R/D : 0, 0  -> add values from depth 0 to 3   (def level 0 always maps to depth 0)
  // incoming R/D : 0, 1  -> add values from depth 0 to 3
  // incoming R/D : 0, 2  -> add values from depth 0 to 3
  // incoming R/D : 1, 4  -> add values from depth 4 to 4
  //
  // Note : the -validity- of values is simply checked by comparing the incoming D value against the
  // D value of the given nesting level (incoming D >= the D for the nesting level == valid,
  // otherwise NULL).  The tricky part is determining what nesting levels to add values at.
  //
  // For schemas with no repetition level (no lists), X is always 0 and Y is always max nesting
  // depth.
  //

  // compute "X" from above
  for (int s_idx = schema.max_repetition_level; s_idx >= 0; s_idx--) {
    auto find_shallowest = [&](int r) {
      int shallowest = -1;
      int cur_depth  = max_depth - 1;
      int schema_idx = src_col_schema;
      while (schema_idx > 0) {
        auto cur_schema = md.get_schema(schema_idx);
        if (cur_schema.max_repetition_level == r) {
          // if this is a repeated field, map it one level deeper
          shallowest = cur_schema.is_stub() ? cur_depth + 1 : cur_depth;
        }
        // if it's one-level encoding list
        else if (cur_schema.is_one_level_list(md.get_schema(cur_schema.parent_idx))) {
          shallowest = cur_depth - 1;
        }
        if (!cur_schema.is_stub()) { cur_depth--; }
        schema_idx = cur_schema.parent_idx;
      }
      return shallowest;
    };
    rep_depth_remap[s_idx] = find_shallowest(s_idx);
  }

  // compute "Y" from above
  for (int s_idx = schema.max_definition_level; s_idx >= 0; s_idx--) {
    auto find_deepest = [&](int d) {
      SchemaElement prev_schema;
      int schema_idx = src_col_schema;
      int r1         = 0;
      while (schema_idx > 0) {
        SchemaElement cur_schema = md.get_schema(schema_idx);
        if (cur_schema.max_definition_level == d) {
          // if this is a repeated field, map it one level deeper
          r1 = cur_schema.is_stub() ? prev_schema.max_repetition_level
                                    : cur_schema.max_repetition_level;
          break;
        }
        prev_schema = cur_schema;
        schema_idx  = cur_schema.parent_idx;
      }

      // we now know R1 from above. return the deepest nesting level that has the
      // same repetition level
      schema_idx = src_col_schema;
      int depth  = max_depth - 1;
      while (schema_idx > 0) {
        SchemaElement cur_schema = md.get_schema(schema_idx);
        if (cur_schema.max_repetition_level == r1) {
          // if this is a repeated field, map it one level deeper
          depth = cur_schema.is_stub() ? depth + 1 : depth;
          break;
        }
        if (!cur_schema.is_stub()) { depth--; }
        prev_schema = cur_schema;
        schema_idx  = cur_schema.parent_idx;
      }
      return depth;
    };
    def_depth_remap[s_idx] = find_deepest(s_idx);
  }
}

/**
 * @brief Reads compressed page data to device memory.
 *
 * @param sources Dataset sources
 * @param page_data Buffers to hold compressed page data for each chunk
 * @param chunks List of column chunk descriptors
 * @param begin_chunk Index of first column chunk to read
 * @param end_chunk Index after the last column chunk to read
 * @param column_chunk_offsets File offset for all chunks
 * @param chunk_source_map Association between each column chunk and its source
 * @param stream CUDA stream used for device memory operations and kernel launches
 *
 * @return A future object for reading synchronization
 */
[[nodiscard]] std::future<void> read_column_chunks_async(
  std::vector<std::unique_ptr<datasource>> const& sources,
  std::vector<std::unique_ptr<datasource::buffer>>& page_data,
  cudf::detail::hostdevice_vector<ColumnChunkDesc>& chunks,
  size_t begin_chunk,
  size_t end_chunk,
  std::vector<size_t> const& column_chunk_offsets,
  std::vector<size_type> const& chunk_source_map,
  rmm::cuda_stream_view stream)
{
  // Transfer chunk data, coalescing adjacent chunks
  std::vector<std::future<size_t>> read_tasks;
  for (size_t chunk = begin_chunk; chunk < end_chunk;) {
    size_t const io_offset   = column_chunk_offsets[chunk];
    size_t io_size           = chunks[chunk].compressed_size;
    size_t next_chunk        = chunk + 1;
    bool const is_compressed = (chunks[chunk].codec != Compression::UNCOMPRESSED);
    while (next_chunk < end_chunk) {
      size_t const next_offset      = column_chunk_offsets[next_chunk];
      bool const is_next_compressed = (chunks[next_chunk].codec != Compression::UNCOMPRESSED);
      if (next_offset != io_offset + io_size || is_next_compressed != is_compressed ||
          chunk_source_map[chunk] != chunk_source_map[next_chunk]) {
        // Can't merge if not contiguous or mixing compressed and uncompressed
        // Not coalescing uncompressed with compressed chunks is so that compressed buffers can be
        // freed earlier (immediately after decompression stage) to limit peak memory requirements
        break;
      }
      io_size += chunks[next_chunk].compressed_size;
      next_chunk++;
    }
    if (io_size != 0) {
      auto& source = sources[chunk_source_map[chunk]];
      if (source->is_device_read_preferred(io_size)) {
        // Buffer needs to be padded.
        // Required by `gpuDecodePageData`.
        auto buffer =
          rmm::device_buffer(cudf::util::round_up_safe(io_size, BUFFER_PADDING_MULTIPLE), stream);
        auto fut_read_size = source->device_read_async(
          io_offset, io_size, static_cast<uint8_t*>(buffer.data()), stream);
        read_tasks.emplace_back(std::move(fut_read_size));
        page_data[chunk] = datasource::buffer::create(std::move(buffer));
      } else {
        auto const read_buffer = source->host_read(io_offset, io_size);
        // Buffer needs to be padded.
        // Required by `gpuDecodePageData`.
        auto tmp_buffer = rmm::device_buffer(
          cudf::util::round_up_safe(read_buffer->size(), BUFFER_PADDING_MULTIPLE), stream);
        CUDF_CUDA_TRY(hipMemcpyAsync(
          tmp_buffer.data(), read_buffer->data(), read_buffer->size(), hipMemcpyDefault, stream));
        page_data[chunk] = datasource::buffer::create(std::move(tmp_buffer));
      }
      auto d_compdata = page_data[chunk]->data();
      do {
        chunks[chunk].compressed_data = d_compdata;
        d_compdata += chunks[chunk].compressed_size;
      } while (++chunk != next_chunk);
    } else {
      chunk = next_chunk;
    }
  }
  auto sync_fn = [](decltype(read_tasks) read_tasks) {
    for (auto& task : read_tasks) {
      task.wait();
    }
  };
  return std::async(std::launch::deferred, sync_fn, std::move(read_tasks));
}

/**
 * @brief Return the number of total pages from the given column chunks.
 *
 * @param chunks List of column chunk descriptors
 * @param stream CUDA stream used for device memory operations and kernel launches
 *
 * @return The total number of pages
 */
[[nodiscard]] size_t count_page_headers(cudf::detail::hostdevice_vector<ColumnChunkDesc>& chunks,
                                        rmm::cuda_stream_view stream)
{
  size_t total_pages = 0;

  kernel_error error_code(stream);
  chunks.host_to_device_async(stream);
  DecodePageHeaders(chunks.device_ptr(), nullptr, chunks.size(), error_code.data(), stream);
  chunks.device_to_host_sync(stream);

  // It's required to ignore unsupported encodings in this function
  // so that we can actually compile a list of all the unsupported encodings found
  // in the pages. That cannot be done here since we do not have the pages vector here.
  // see https://github.com/rapidsai/cudf/pull/14453#pullrequestreview-1778346688
  if (auto const error = error_code.value_sync(stream);
      error != 0 and error != static_cast<uint32_t>(decode_error::UNSUPPORTED_ENCODING)) {
    CUDF_FAIL("Parquet header parsing failed with code(s) while counting page headers " +
              kernel_error::to_string(error));
  }

  for (size_t c = 0; c < chunks.size(); c++) {
    total_pages += chunks[c].num_data_pages + chunks[c].num_dict_pages;
  }

  return total_pages;
}

/**
 * @brief Count the total number of pages using page index information.
 */
[[nodiscard]] size_t count_page_headers_with_pgidx(
  cudf::detail::hostdevice_vector<ColumnChunkDesc>& chunks, rmm::cuda_stream_view stream)
{
  size_t total_pages = 0;
  for (auto& chunk : chunks) {
    CUDF_EXPECTS(chunk.h_chunk_info != nullptr, "Expected non-null column info struct");
    auto const& chunk_info = *chunk.h_chunk_info;
    chunk.num_dict_pages   = chunk_info.has_dictionary() ? 1 : 0;
    chunk.num_data_pages   = chunk_info.pages.size();
    total_pages += chunk.num_data_pages + chunk.num_dict_pages;
  }

  // count_page_headers() also pushes chunks to device, so not using thrust here
  chunks.host_to_device_async(stream);

  return total_pages;
}

// struct used to carry info from the page indexes to the device
struct page_index_info {
  int32_t num_rows;
  int32_t chunk_row;
  int32_t num_nulls;
  int32_t num_valids;
  int32_t str_bytes;
};

// functor to copy page_index_info into the PageInfo struct
struct copy_page_info {
  device_span<page_index_info const> page_indexes;
  device_span<PageInfo> pages;

  __device__ void operator()(size_type idx)
  {
    auto& pg                = pages[idx];
    auto const& pi          = page_indexes[idx];
    pg.num_rows             = pi.num_rows;
    pg.chunk_row            = pi.chunk_row;
    pg.has_page_index       = true;
    pg.num_nulls            = pi.num_nulls;
    pg.num_valids           = pi.num_valids;
    pg.str_bytes_from_index = pi.str_bytes;
    pg.str_bytes            = pi.str_bytes;
    pg.start_val            = 0;
    pg.end_val              = pg.num_valids;
  }
};

/**
 * @brief Set fields on the pages that can be derived from page indexes.
 *
 * This replaces some preprocessing steps, such as page string size calculation.
 */
void fill_in_page_info(host_span<ColumnChunkDesc> chunks,
                       device_span<PageInfo> pages,
                       rmm::cuda_stream_view stream)
{
  auto const num_pages = pages.size();
  std::vector<page_index_info> page_indexes(num_pages);

  for (size_t c = 0, page_count = 0; c < chunks.size(); c++) {
    auto const& chunk = chunks[c];
    CUDF_EXPECTS(chunk.h_chunk_info != nullptr, "Expected non-null column info struct");
    auto const& chunk_info = *chunk.h_chunk_info;
    size_t start_row       = 0;
    page_count += chunk.num_dict_pages;
    for (size_t p = 0; p < chunk_info.pages.size(); p++, page_count++) {
      auto& page      = page_indexes[page_count];
      page.num_rows   = chunk_info.pages[p].num_rows;
      page.chunk_row  = start_row;
      page.num_nulls  = chunk_info.pages[p].num_nulls.value_or(0);
      page.num_valids = chunk_info.pages[p].num_valid.value_or(0);
      page.str_bytes  = chunk_info.pages[p].var_bytes_size.value_or(0);

      start_row += page.num_rows;
    }
  }

  auto d_page_indexes = cudf::detail::make_device_uvector_async(
    page_indexes, stream, rmm::mr::get_current_device_resource());

  auto iter = thrust::make_counting_iterator<size_type>(0);
  thrust::for_each(
    rmm::exec_policy_nosync(stream), iter, iter + num_pages, copy_page_info{d_page_indexes, pages});
}

/**
 * @brief Returns a string representation of known encodings
 *
 * @param encoding Given encoding
 * @return String representation of encoding
 */
std::string encoding_to_string(Encoding encoding)
{
  switch (encoding) {
    case Encoding::PLAIN: return "PLAIN";
    case Encoding::GROUP_VAR_INT: return "GROUP_VAR_INT";
    case Encoding::PLAIN_DICTIONARY: return "PLAIN_DICTIONARY";
    case Encoding::RLE: return "RLE";
    case Encoding::BIT_PACKED: return "BIT_PACKED";
    case Encoding::DELTA_BINARY_PACKED: return "DELTA_BINARY_PACKED";
    case Encoding::DELTA_LENGTH_BYTE_ARRAY: return "DELTA_LENGTH_BYTE_ARRAY";
    case Encoding::DELTA_BYTE_ARRAY: return "DELTA_BYTE_ARRAY";
    case Encoding::RLE_DICTIONARY: return "RLE_DICTIONARY";
    case Encoding::BYTE_STREAM_SPLIT: return "BYTE_STREAM_SPLIT";
    case Encoding::NUM_ENCODINGS:
    default: return "UNKNOWN(" + std::to_string(static_cast<int>(encoding)) + ")";
  }
}

/**
 * @brief Helper function to convert an encoding bitmask to a readable string
 *
 * @param bitmask Bitmask of found unsupported encodings
 * @returns Human readable string with unsupported encodings
 */
[[nodiscard]] std::string encoding_bitmask_to_str(uint32_t encoding_bitmask)
{
  std::bitset<32> bits(encoding_bitmask);
  std::string result;

  for (size_t i = 0; i < bits.size(); ++i) {
    if (bits.test(i)) {
      auto const current = static_cast<Encoding>(i);
      if (!is_supported_encoding(current)) { result.append(encoding_to_string(current) + " "); }
    }
  }
  return result;
}

/**
 * @brief Create a readable string for the user that will list out all unsupported encodings found.
 *
 * @param pages List of page information
 * @param stream CUDA stream used for device memory operations and kernel launches
 * @returns Human readable string with unsupported encodings
 */
[[nodiscard]] std::string list_unsupported_encodings(device_span<PageInfo const> pages,
                                                     rmm::cuda_stream_view stream)
{
  auto const to_mask = [] __device__(auto const& page) {
    return is_supported_encoding(page.encoding) ? 0U : encoding_to_mask(page.encoding);
  };
  uint32_t const unsupported = thrust::transform_reduce(
    rmm::exec_policy(stream), pages.begin(), pages.end(), to_mask, 0U, thrust::bit_or<uint32_t>());
  return encoding_bitmask_to_str(unsupported);
}

/**
 * @brief Sort pages in chunk/schema order
 *
 * @param unsorted_pages The unsorted pages
 * @param chunks The chunks associated with the pages
 * @param stream CUDA stream used for device memory operations and kernel launches
 * @returns The sorted vector of pages
 */
cudf::detail::hostdevice_vector<PageInfo> sort_pages(device_span<PageInfo const> unsorted_pages,
                                                     device_span<ColumnChunkDesc const> chunks,
                                                     rmm::cuda_stream_view stream)
{
  CUDF_FUNC_RANGE();

  // sort the pages in chunk/schema order. we use chunk.src_col_index instead of
  // chunk.src_col_schema because the user may have reordered them (reading columns, "a" and "b" but
  // returning them as "b" and "a")
  //
  // ordering of pages is by input column schema, repeated across row groups.  so
  // if we had 3 columns, each with 2 pages, and 1 row group, our schema values might look like
  //
  // 1, 1, 2, 2, 3, 3
  //
  // However, if we had more than one row group, the pattern would be
  //
  // 1, 1, 2, 2, 3, 3, 1, 1, 2, 2, 3, 3
  // ^ row group 0     |
  //                   ^ row group 1
  //
  // To process pages by key (exclusive_scan_by_key, reduce_by_key, etc), the ordering we actually
  // want is
  //
  // 1, 1, 1, 1, 2, 2, 2, 2, 3, 3, 3, 3
  //
  // We also need to preserve key-relative page ordering, so we need to use a stable sort.
  rmm::device_uvector<int32_t> page_keys{unsorted_pages.size(), stream};
  thrust::transform(
    rmm::exec_policy_nosync(stream),
    unsorted_pages.begin(),
    unsorted_pages.end(),
    page_keys.begin(),
    cuda::proclaim_return_type<int32_t>([chunks = chunks.begin()] __device__(PageInfo const& page) {
      return chunks[page.chunk_idx].src_col_index;
    }));
  // we are doing this by sorting indices first and then transforming the output because nvcc
  // started generating kernels using too much shared memory when trying to sort the pages
  // directly.
  rmm::device_uvector<int32_t> sort_indices(unsorted_pages.size(), stream);
  thrust::sequence(rmm::exec_policy_nosync(stream), sort_indices.begin(), sort_indices.end(), 0);
  thrust::stable_sort_by_key(rmm::exec_policy_nosync(stream),
                             page_keys.begin(),
                             page_keys.end(),
                             sort_indices.begin(),
                             thrust::less<int>());
  auto pass_pages =
    cudf::detail::hostdevice_vector<PageInfo>(unsorted_pages.size(), unsorted_pages.size(), stream);
  thrust::transform(
    rmm::exec_policy_nosync(stream),
    sort_indices.begin(),
    sort_indices.end(),
    pass_pages.d_begin(),
    cuda::proclaim_return_type<PageInfo>([unsorted_pages = unsorted_pages.begin()] __device__(
                                           int32_t i) { return unsorted_pages[i]; }));
  stream.synchronize();
  return pass_pages;
}

/**
 * @brief Decode the page information for a given pass.
 *
 * @param pass_intermediate_data The struct containing pass information
 */
void decode_page_headers(pass_intermediate_data& pass,
                         device_span<PageInfo> unsorted_pages,
                         bool has_page_index,
                         rmm::cuda_stream_view stream)
{
  CUDF_FUNC_RANGE();

  auto iter = thrust::make_counting_iterator(0);
  rmm::device_uvector<size_t> chunk_page_counts(pass.chunks.size() + 1, stream);
  thrust::transform_exclusive_scan(
    rmm::exec_policy_nosync(stream),
    iter,
    iter + pass.chunks.size() + 1,
    chunk_page_counts.begin(),
    cuda::proclaim_return_type<size_t>(
      [chunks = pass.chunks.d_begin(), num_chunks = pass.chunks.size()] __device__(size_t i) {
        return static_cast<size_t>(
          i >= num_chunks ? 0 : chunks[i].num_data_pages + chunks[i].num_dict_pages);
      }),
    0,
    thrust::plus<size_t>{});
  rmm::device_uvector<chunk_page_info> d_chunk_page_info(pass.chunks.size(), stream);
  thrust::for_each(rmm::exec_policy_nosync(stream),
                   iter,
                   iter + pass.chunks.size(),
                   [cpi               = d_chunk_page_info.begin(),
                    chunk_page_counts = chunk_page_counts.begin(),
                    unsorted_pages    = unsorted_pages.begin()] __device__(size_t i) {
                     cpi[i].pages = &unsorted_pages[chunk_page_counts[i]];
                   });

  kernel_error error_code(stream);
  DecodePageHeaders(pass.chunks.d_begin(),
                    d_chunk_page_info.begin(),
                    pass.chunks.size(),
                    error_code.data(),
                    stream);

  if (auto const error = error_code.value_sync(stream); error != 0) {
    if (BitAnd(error, decode_error::UNSUPPORTED_ENCODING) != 0) {
      auto const unsupported_str =
        ". With unsupported encodings found: " + list_unsupported_encodings(pass.pages, stream);
      CUDF_FAIL("Parquet header parsing failed with code(s) " + kernel_error::to_string(error) +
                unsupported_str);
    } else {
      CUDF_FAIL("Parquet header parsing failed with code(s) " + kernel_error::to_string(error));
    }
  }

  if (has_page_index) { fill_in_page_info(pass.chunks, unsorted_pages, stream); }

  // compute max bytes needed for level data
  auto level_bit_size = cudf::detail::make_counting_transform_iterator(
    0, cuda::proclaim_return_type<int>([chunks = pass.chunks.d_begin()] __device__(int i) {
      auto c = chunks[i];
      return static_cast<int>(
        max(c.level_bits[level_type::REPETITION], c.level_bits[level_type::DEFINITION]));
    }));
  // max level data bit size.
  int const max_level_bits = thrust::reduce(rmm::exec_policy(stream),
                                            level_bit_size,
                                            level_bit_size + pass.chunks.size(),
                                            0,
                                            thrust::maximum<int>());
  pass.level_type_size     = std::max(1, cudf::util::div_rounding_up_safe(max_level_bits, 8));

  // sort the pages in chunk/schema order.
  pass.pages = sort_pages(unsorted_pages, pass.chunks, stream);

  // compute offsets to each group of input pages.
  // page_keys:   1, 1, 1, 1, 2, 2, 2, 2, 3, 3, 3, 3
  //
  // result:      0,          4,          8
  rmm::device_uvector<size_type> page_counts(pass.pages.size() + 1, stream);
  auto page_keys             = make_page_key_iterator(pass.pages);
  auto const page_counts_end = thrust::reduce_by_key(rmm::exec_policy(stream),
                                                     page_keys,
                                                     page_keys + pass.pages.size(),
                                                     thrust::make_constant_iterator(1),
                                                     thrust::make_discard_iterator(),
                                                     page_counts.begin())
                                 .second;
  auto const num_page_counts = page_counts_end - page_counts.begin();
  pass.page_offsets          = rmm::device_uvector<size_type>(num_page_counts + 1, stream);
  thrust::exclusive_scan(rmm::exec_policy_nosync(stream),
                         page_counts.begin(),
                         page_counts.begin() + num_page_counts + 1,
                         pass.page_offsets.begin());

  // setup dict_page for each chunk if necessary
  thrust::for_each(rmm::exec_policy_nosync(stream),
                   pass.pages.d_begin(),
                   pass.pages.d_end(),
                   [chunks = pass.chunks.d_begin()] __device__(PageInfo const& p) {
                     if (p.flags & PAGEINFO_FLAGS_DICTIONARY) {
                       chunks[p.chunk_idx].dict_page = &p;
                     }
                   });

  pass.pages.device_to_host_async(stream);
  pass.chunks.device_to_host_async(stream);
  stream.synchronize();
}

constexpr bool is_string_chunk(ColumnChunkDesc const& chunk)
{
  auto const is_decimal =
    chunk.logical_type.has_value() and chunk.logical_type->type == LogicalType::DECIMAL;
  auto const is_binary =
    chunk.physical_type == BYTE_ARRAY or chunk.physical_type == FIXED_LEN_BYTE_ARRAY;
  return is_binary and not is_decimal;
}

struct set_str_dict_index_count {
  device_span<size_t> str_dict_index_count;
  device_span<const ColumnChunkDesc> chunks;

  __device__ void operator()(PageInfo const& page)
  {
    auto const& chunk = chunks[page.chunk_idx];
    if ((page.flags & PAGEINFO_FLAGS_DICTIONARY) != 0 and chunk.num_dict_pages > 0 and
        is_string_chunk(chunk)) {
      // there is only ever one dictionary page per chunk, so this is safe to do in parallel.
      str_dict_index_count[page.chunk_idx] = page.num_input_values;
    }
  }
};

struct set_str_dict_index_ptr {
  string_index_pair* const base;
  device_span<const size_t> str_dict_index_offsets;
  device_span<ColumnChunkDesc> chunks;

  __device__ void operator()(size_t i)
  {
    auto& chunk = chunks[i];
    if (chunk.num_dict_pages > 0 and is_string_chunk(chunk)) {
      chunk.str_dict_index = base + str_dict_index_offsets[i];
    }
  }
};

/**
 * @brief Functor which computes an estimated row count for list pages.
 *
 */
struct set_list_row_count_estimate {
  device_span<const ColumnChunkDesc> chunks;

  __device__ void operator()(PageInfo& page)
  {
    if (page.flags & PAGEINFO_FLAGS_DICTIONARY) { return; }
    auto const& chunk  = chunks[page.chunk_idx];
    auto const is_list = chunk.max_level[level_type::REPETITION] > 0;
    if (!is_list) { return; }

    // For LIST pages that we have not yet decoded, page.num_rows is not an accurate number.
    // so we instead estimate the number of rows as follows:
    // - each chunk stores an estimated number of bytes per row E
    // - estimate number of rows in a page = page.uncompressed_page_size / E
    //
    // it is not required that this number is accurate. we just want it to be somewhat close so that
    // we get reasonable results as we choose subpass splits.
    //
    // all other columns can use page.num_rows directly as it will be accurate.
    page.num_rows = static_cast<size_t>(static_cast<float>(page.uncompressed_page_size) /
                                        chunk.list_bytes_per_row_est);
  }
};

/**
 * @brief Set the expected row count on the final page for all columns.
 *
 */
struct set_final_row_count {
  device_span<PageInfo> pages;
  device_span<const ColumnChunkDesc> chunks;

  __device__ void operator()(size_t i)
  {
    auto& page        = pages[i];
    auto const& chunk = chunks[page.chunk_idx];
    // only do this for the last page in each chunk
    if (i < pages.size() - 1 && (pages[i + 1].chunk_idx == page.chunk_idx)) { return; }
    size_t const page_start_row = chunk.start_row + page.chunk_row;
    size_t const chunk_last_row = chunk.start_row + chunk.num_rows;
    page.num_rows               = chunk_last_row - page_start_row;
  }
};

}  // anonymous namespace

void reader::impl::build_string_dict_indices()
{
  CUDF_FUNC_RANGE();

  auto& pass = *_pass_itm_data;

  // compute number of indices per chunk and a summed total
  rmm::device_uvector<size_t> str_dict_index_count(pass.chunks.size() + 1, _stream);
  thrust::fill(
    rmm::exec_policy_nosync(_stream), str_dict_index_count.begin(), str_dict_index_count.end(), 0);
  thrust::for_each(rmm::exec_policy_nosync(_stream),
                   pass.pages.d_begin(),
                   pass.pages.d_end(),
                   set_str_dict_index_count{str_dict_index_count, pass.chunks});

  size_t const total_str_dict_indexes = thrust::reduce(
    rmm::exec_policy(_stream), str_dict_index_count.begin(), str_dict_index_count.end());
  if (total_str_dict_indexes == 0) { return; }

  // convert to offsets
  rmm::device_uvector<size_t>& str_dict_index_offsets = str_dict_index_count;
  thrust::exclusive_scan(rmm::exec_policy_nosync(_stream),
                         str_dict_index_offsets.begin(),
                         str_dict_index_offsets.end(),
                         str_dict_index_offsets.begin(),
                         0);

  // allocate and distribute pointers
  pass.str_dict_index = cudf::detail::make_zeroed_device_uvector_async<string_index_pair>(
    total_str_dict_indexes, _stream, rmm::mr::get_current_device_resource());

  auto iter = thrust::make_counting_iterator(0);
  thrust::for_each(
    rmm::exec_policy_nosync(_stream),
    iter,
    iter + pass.chunks.size(),
    set_str_dict_index_ptr{pass.str_dict_index.data(), str_dict_index_offsets, pass.chunks});

  // compute the indices
  BuildStringDictionaryIndex(pass.chunks.device_ptr(), pass.chunks.size(), _stream);
  pass.chunks.device_to_host_sync(_stream);
}

void reader::impl::allocate_nesting_info()
{
  auto& pass    = *_pass_itm_data;
  auto& subpass = *pass.subpass;

  auto const num_columns         = _input_columns.size();
  auto& pages                    = subpass.pages;
  auto& page_nesting_info        = subpass.page_nesting_info;
  auto& page_nesting_decode_info = subpass.page_nesting_decode_info;

  // generate the number of nesting info structs needed per-page, by column
  std::vector<int> per_page_nesting_info_size(num_columns);
  auto iter = thrust::make_counting_iterator(size_type{0});
  std::transform(iter, iter + num_columns, per_page_nesting_info_size.begin(), [&](size_type i) {
    auto const schema_idx = _input_columns[i].schema_idx;
    auto const& schema    = _metadata->get_schema(schema_idx);
    return max(schema.max_definition_level + 1, _metadata->get_output_nesting_depth(schema_idx));
  });

  // compute total # of page_nesting infos needed and allocate space. doing this in one
  // buffer to keep it to a single gpu allocation
  auto counting_iter = thrust::make_counting_iterator(size_t{0});
  size_t const total_page_nesting_infos =
    std::accumulate(counting_iter, counting_iter + num_columns, 0, [&](int total, size_t index) {
      return total + (per_page_nesting_info_size[index] * subpass.column_page_count[index]);
    });

  page_nesting_info =
    cudf::detail::hostdevice_vector<PageNestingInfo>{total_page_nesting_infos, _stream};
  page_nesting_decode_info =
    cudf::detail::hostdevice_vector<PageNestingDecodeInfo>{total_page_nesting_infos, _stream};

  // update pointers in the PageInfos
  int target_page_index = 0;
  int src_info_index    = 0;
  for (size_t idx = 0; idx < _input_columns.size(); idx++) {
    auto const src_col_schema = _input_columns[idx].schema_idx;

    for (size_t p_idx = 0; p_idx < subpass.column_page_count[idx]; p_idx++) {
      pages[target_page_index + p_idx].nesting = page_nesting_info.device_ptr() + src_info_index;
      pages[target_page_index + p_idx].nesting_decode =
        page_nesting_decode_info.device_ptr() + src_info_index;

      pages[target_page_index + p_idx].nesting_info_size = per_page_nesting_info_size[idx];
      pages[target_page_index + p_idx].num_output_nesting_levels =
        _metadata->get_output_nesting_depth(src_col_schema);

      src_info_index += per_page_nesting_info_size[idx];
    }
    target_page_index += subpass.column_page_count[idx];
  }

  // fill in
  int nesting_info_index = 0;
  std::map<int, std::pair<std::vector<int>, std::vector<int>>> depth_remapping;
  for (size_t idx = 0; idx < _input_columns.size(); idx++) {
    auto const src_col_schema = _input_columns[idx].schema_idx;

    // schema of the input column
    auto& schema = _metadata->get_schema(src_col_schema);
    // real depth of the output cudf column hierarchy (1 == no nesting, 2 == 1 level, etc)
    int const max_output_depth = _metadata->get_output_nesting_depth(src_col_schema);

    // if this column has lists, generate depth remapping
    std::map<int, std::pair<std::vector<int>, std::vector<int>>> depth_remapping;
    if (schema.max_repetition_level > 0) {
      generate_depth_remappings(depth_remapping, src_col_schema, *_metadata);
    }

    // fill in host-side nesting info
    int schema_idx  = src_col_schema;
    auto cur_schema = _metadata->get_schema(schema_idx);
    int cur_depth   = max_output_depth - 1;
    while (schema_idx > 0) {
      // stub columns (basically the inner field of a list schema element) are not real columns.
      // we can ignore them for the purposes of output nesting info
      if (!cur_schema.is_stub()) {
        // initialize each page within the chunk
        for (size_t p_idx = 0; p_idx < subpass.column_page_count[idx]; p_idx++) {
          PageNestingInfo* pni =
            &page_nesting_info[nesting_info_index + (p_idx * per_page_nesting_info_size[idx])];

          PageNestingDecodeInfo* nesting_info =
            &page_nesting_decode_info[nesting_info_index +
                                      (p_idx * per_page_nesting_info_size[idx])];

          // if we have lists, set our start and end depth remappings
          if (schema.max_repetition_level > 0) {
            auto remap = depth_remapping.find(src_col_schema);
            CUDF_EXPECTS(remap != depth_remapping.end(),
                         "Could not find depth remapping for schema");
            std::vector<int> const& rep_depth_remap = (remap->second.first);
            std::vector<int> const& def_depth_remap = (remap->second.second);

            for (size_t m = 0; m < rep_depth_remap.size(); m++) {
              nesting_info[m].start_depth = rep_depth_remap[m];
            }
            for (size_t m = 0; m < def_depth_remap.size(); m++) {
              nesting_info[m].end_depth = def_depth_remap[m];
            }
          }

          // values indexed by output column index
          nesting_info[cur_depth].max_def_level = cur_schema.max_definition_level;
          pni[cur_depth].size                   = 0;
          pni[cur_depth].type =
            to_type_id(cur_schema, _strings_to_categorical, _timestamp_type.id());
          pni[cur_depth].nullable = cur_schema.repetition_type == OPTIONAL;
        }

        // move up the hierarchy
        cur_depth--;
      }

      // next schema
      schema_idx = cur_schema.parent_idx;
      cur_schema = _metadata->get_schema(schema_idx);
    }

    nesting_info_index += (per_page_nesting_info_size[idx] * subpass.column_page_count[idx]);
  }

  // copy nesting info to the device
  page_nesting_info.host_to_device_async(_stream);
  page_nesting_decode_info.host_to_device_async(_stream);
}

void reader::impl::allocate_level_decode_space()
{
  auto& pass    = *_pass_itm_data;
  auto& subpass = *pass.subpass;

  auto& pages = subpass.pages;

  // TODO: this could be made smaller if we ignored dictionary pages and pages with no
  // repetition data.
  size_t const per_page_decode_buf_size = LEVEL_DECODE_BUF_SIZE * 2 * pass.level_type_size;
  auto const decode_buf_size            = per_page_decode_buf_size * pages.size();
  subpass.level_decode_data =
    rmm::device_buffer(decode_buf_size, _stream, rmm::mr::get_current_device_resource());

  // distribute the buffers
  uint8_t* buf = static_cast<uint8_t*>(subpass.level_decode_data.data());
  for (size_t idx = 0; idx < pages.size(); idx++) {
    auto& p = pages[idx];

    p.lvl_decode_buf[level_type::DEFINITION] = buf;
    buf += (LEVEL_DECODE_BUF_SIZE * pass.level_type_size);
    p.lvl_decode_buf[level_type::REPETITION] = buf;
    buf += (LEVEL_DECODE_BUF_SIZE * pass.level_type_size);
  }
}

std::pair<bool, std::vector<std::future<void>>> reader::impl::read_column_chunks()
{
  auto const& row_groups_info = _pass_itm_data->row_groups;

  auto& raw_page_data = _pass_itm_data->raw_page_data;
  auto& chunks        = _pass_itm_data->chunks;

  // Descriptors for all the chunks that make up the selected columns
  auto const num_input_columns = _input_columns.size();
  auto const num_chunks        = row_groups_info.size() * num_input_columns;

  // Association between each column chunk and its source
  std::vector<size_type> chunk_source_map(num_chunks);

  // Tracker for eventually deallocating compressed and uncompressed data
  raw_page_data = std::vector<std::unique_ptr<datasource::buffer>>(num_chunks);

  // Keep track of column chunk file offsets
  std::vector<size_t> column_chunk_offsets(num_chunks);

  // Initialize column chunk information
  size_t total_decompressed_size = 0;
  // TODO: make this respect the pass-wide skip_rows/num_rows instead of the file-wide
  // skip_rows/num_rows
  // auto remaining_rows            = num_rows;
  std::vector<std::future<void>> read_chunk_tasks;
  size_type chunk_count = 0;
  for (auto const& rg : row_groups_info) {
    auto const& row_group       = _metadata->get_row_group(rg.index, rg.source_index);
    auto const row_group_source = rg.source_index;

    // generate ColumnChunkDesc objects for everything to be decoded (all input columns)
    for (size_t i = 0; i < num_input_columns; ++i) {
      auto const& col = _input_columns[i];
      // look up metadata
      auto& col_meta = _metadata->get_column_metadata(rg.index, rg.source_index, col.schema_idx);

      column_chunk_offsets[chunk_count] =
        (col_meta.dictionary_page_offset != 0)
          ? std::min(col_meta.data_page_offset, col_meta.dictionary_page_offset)
          : col_meta.data_page_offset;

      // Map each column chunk to its column index and its source index
      chunk_source_map[chunk_count] = row_group_source;

      if (col_meta.codec != Compression::UNCOMPRESSED) {
        total_decompressed_size += col_meta.total_uncompressed_size;
      }

      chunk_count++;
    }
  }

  // Read compressed chunk data to device memory
  read_chunk_tasks.push_back(read_column_chunks_async(_sources,
                                                      raw_page_data,
                                                      chunks,
                                                      0,
                                                      chunks.size(),
                                                      column_chunk_offsets,
                                                      chunk_source_map,
                                                      _stream));

  return {total_decompressed_size > 0, std::move(read_chunk_tasks)};
}

void reader::impl::read_compressed_data()
{
  auto& pass = *_pass_itm_data;

  // This function should never be called if `num_rows == 0`.
  CUDF_EXPECTS(_pass_itm_data->num_rows > 0, "Number of reading rows must not be zero.");

  auto& chunks = pass.chunks;

  auto const [has_compressed_data, read_chunks_tasks] = read_column_chunks();
  pass.has_compressed_data                            = has_compressed_data;

  for (auto& task : read_chunks_tasks) {
    task.wait();
  }

  // Process dataset chunk pages into output columns
  auto const total_pages = _has_page_index ? count_page_headers_with_pgidx(chunks, _stream)
                                           : count_page_headers(chunks, _stream);
  if (total_pages <= 0) { return; }
  rmm::device_uvector<PageInfo> unsorted_pages(total_pages, _stream);

  // decoding of column/page information
  decode_page_headers(pass, unsorted_pages, _has_page_index, _stream);
  CUDF_EXPECTS(pass.page_offsets.size() - 1 == static_cast<size_t>(_input_columns.size()),
               "Encountered page_offsets / num_columns mismatch");
}

namespace {

struct cumulative_row_info {
  size_t row_count;   // cumulative row count
  size_t size_bytes;  // cumulative size in bytes
  int key;            // schema index
};

struct get_page_chunk_idx {
  __device__ size_type operator()(PageInfo const& page) { return page.chunk_idx; }
};

struct get_page_num_rows {
  __device__ size_type operator()(PageInfo const& page) { return page.num_rows; }
};

struct input_col_info {
  int const schema_idx;
  size_type const nesting_depth;
};

/**
 * @brief Converts a 1-dimensional index into page, depth and column indices used in
 * allocate_columns to compute columns sizes.
 *
 * The input index will iterate through pages, nesting depth and column indices in that order.
 */
struct reduction_indices {
  size_t const page_idx;
  size_type const depth_idx;
  size_type const col_idx;

  __device__ reduction_indices(size_t index_, size_type max_depth_, size_t num_pages_)
    : page_idx(index_ % num_pages_),
      depth_idx((index_ / num_pages_) % max_depth_),
      col_idx(index_ / (max_depth_ * num_pages_))
  {
  }
};

/**
 * @brief Returns the size field of a PageInfo struct for a given depth, keyed by schema.
 */
struct get_page_nesting_size {
  input_col_info const* const input_cols;
  size_type const max_depth;
  size_t const num_pages;
  PageInfo const* const pages;

  __device__ size_type operator()(size_t index) const
  {
    auto const indices = reduction_indices{index, max_depth, num_pages};

    auto const& page = pages[indices.page_idx];
    if (page.src_col_schema != input_cols[indices.col_idx].schema_idx ||
        page.flags & PAGEINFO_FLAGS_DICTIONARY ||
        indices.depth_idx >= input_cols[indices.col_idx].nesting_depth) {
      return 0;
    }

    return page.nesting[indices.depth_idx].batch_size;
  }
};

struct get_reduction_key {
  size_t const num_pages;
  __device__ size_t operator()(size_t index) const { return index / num_pages; }
};

/**
 * @brief Writes to the chunk_row field of the PageInfo struct.
 */
struct chunk_row_output_iter {
  PageInfo* p;
  using value_type        = size_type;
  using difference_type   = size_type;
  using pointer           = size_type*;
  using reference         = size_type&;
  using iterator_category = thrust::output_device_iterator_tag;

  __host__ __device__ chunk_row_output_iter operator+(int i) { return {p + i}; }

  __host__ __device__ chunk_row_output_iter& operator++()
  {
    p++;
    return *this;
  }

  __device__ reference operator[](int i) { return p[i].chunk_row; }
  __device__ reference operator*() { return p->chunk_row; }
};

/**
 * @brief Writes to the page_start_value field of the PageNestingInfo struct, keyed by schema.
 */
/**
 * @brief Writes to the page_start_value field of the PageNestingInfo struct, keyed by schema.
 */
struct start_offset_output_iterator {
  PageInfo const* pages;
  size_t cur_index;
  input_col_info const* input_cols;
  size_type max_depth;
  size_t num_pages;
  int empty               = 0;
  using value_type        = size_type;
  using difference_type   = size_type;
  using pointer           = size_type*;
  using reference         = size_type&;
  using iterator_category = thrust::output_device_iterator_tag;

  constexpr void operator=(start_offset_output_iterator const& other)
  {
    pages      = other.pages;
    cur_index  = other.cur_index;
    input_cols = other.input_cols;
    max_depth  = other.max_depth;
    num_pages  = other.num_pages;
  }

  constexpr start_offset_output_iterator operator+(size_t i)
  {
    return start_offset_output_iterator{pages, cur_index + i, input_cols, max_depth, num_pages};
  }

  constexpr start_offset_output_iterator& operator++()
  {
    cur_index++;
    return *this;
  }

  __device__ reference operator[](size_t i) { return dereference(cur_index + i); }
  __device__ reference operator*() { return dereference(cur_index); }

 private:
  __device__ reference dereference(size_t index)
  {
    auto const indices = reduction_indices{index, max_depth, num_pages};

    PageInfo const& p = pages[indices.page_idx];
    if (p.src_col_schema != input_cols[indices.col_idx].schema_idx ||
        p.flags & PAGEINFO_FLAGS_DICTIONARY ||
        indices.depth_idx >= input_cols[indices.col_idx].nesting_depth) {
      return empty;
    }
    return p.nesting_decode[indices.depth_idx].page_start_value;
  }
};

struct page_to_string_size {
  ColumnChunkDesc const* chunks;

  __device__ size_t operator()(PageInfo const& page) const
  {
    auto const chunk = chunks[page.chunk_idx];

    if (not is_string_col(chunk) || (page.flags & PAGEINFO_FLAGS_DICTIONARY) != 0) { return 0; }
    return page.str_bytes;
  }
};

struct page_offset_output_iter {
  PageInfo* p;

  using value_type        = size_t;
  using difference_type   = size_t;
  using pointer           = size_t*;
  using reference         = size_t&;
  using iterator_category = thrust::output_device_iterator_tag;

  __host__ __device__ page_offset_output_iter operator+(int i) { return {p + i}; }

  __host__ __device__ page_offset_output_iter& operator++()
  {
    p++;
    return *this;
  }

  __device__ reference operator[](int i) { return p[i].str_offset; }
  __device__ reference operator*() { return p->str_offset; }
};
// update chunk_row field in subpass page from pass page
struct update_subpass_chunk_row {
  device_span<PageInfo> pass_pages;
  device_span<PageInfo> subpass_pages;
  device_span<size_t> page_src_index;

  __device__ void operator()(size_t i)
  {
    subpass_pages[i].chunk_row = pass_pages[page_src_index[i]].chunk_row;
  }
};

// update num_rows field from pass page to subpass page
struct update_pass_num_rows {
  device_span<PageInfo> pass_pages;
  device_span<PageInfo> subpass_pages;
  device_span<size_t> page_src_index;

  __device__ void operator()(size_t i)
  {
    pass_pages[page_src_index[i]].num_rows = subpass_pages[i].num_rows;
  }
};

}  // anonymous namespace

void reader::impl::preprocess_file(
  int64_t skip_rows,
  std::optional<size_type> const& num_rows,
  host_span<std::vector<size_type> const> row_group_indices,
  std::optional<std::reference_wrapper<ast::expression const>> filter)
{
  CUDF_EXPECTS(!_file_preprocessed, "Attempted to preprocess file more than once");

  // if filter is not empty, then create output types as vector and pass for filtering.
  std::vector<data_type> output_types;
  if (filter.has_value()) {
    std::transform(_output_buffers.cbegin(),
                   _output_buffers.cend(),
                   std::back_inserter(output_types),
                   [](auto const& col) { return col.type; });
  }
  std::tie(
    _file_itm_data.global_skip_rows, _file_itm_data.global_num_rows, _file_itm_data.row_groups) =
    _metadata->select_row_groups(
      row_group_indices, skip_rows, num_rows, output_types, filter, _stream);

  // check for page indexes
  _has_page_index = std::all_of(_file_itm_data.row_groups.begin(),
                                _file_itm_data.row_groups.end(),
                                [](auto const& row_group) { return row_group.has_page_index(); });

  if (_file_itm_data.global_num_rows > 0 && not _file_itm_data.row_groups.empty() &&
      not _input_columns.empty()) {
    // fills in chunk information without physically loading or decompressing
    // the associated data
    create_global_chunk_info();

    // compute schedule of input reads.
    compute_input_passes();
  }

#if defined(PARQUET_CHUNK_LOGGING)
  printf("==============================================\n");
  setlocale(LC_NUMERIC, "");
  printf("File: skip_rows(%'lu), num_rows(%'lu), input_read_limit(%'lu), output_read_limit(%'lu)\n",
         _file_itm_data.global_skip_rows,
         _file_itm_data.global_num_rows,
         _input_pass_read_limit,
         _output_chunk_read_limit);
  printf("# Row groups: %'lu\n", _file_itm_data.row_groups.size());
  printf("# Input passes: %'lu\n", _file_itm_data.num_passes());
  printf("# Input columns: %'lu\n", _input_columns.size());
  for (size_t idx = 0; idx < _input_columns.size(); idx++) {
    auto const& schema = _metadata->get_schema(_input_columns[idx].schema_idx);
    auto const type_id = to_type_id(schema, _strings_to_categorical, _timestamp_type.id());
    printf("\tC(%'lu, %s): %s\n",
           idx,
           _input_columns[idx].name.c_str(),
           cudf::type_to_name(cudf::data_type{type_id}).c_str());
  }
  printf("# Output columns: %'lu\n", _output_buffers.size());
  for (size_t idx = 0; idx < _output_buffers.size(); idx++) {
    printf("\tC(%'lu): %s\n", idx, cudf::io::detail::type_to_name(_output_buffers[idx]).c_str());
  }
#endif

  _file_preprocessed = true;
}

void reader::impl::generate_list_column_row_count_estimates()
{
  auto& pass = *_pass_itm_data;
  thrust::for_each(rmm::exec_policy(_stream),
                   pass.pages.d_begin(),
                   pass.pages.d_end(),
                   set_list_row_count_estimate{pass.chunks});

  // computes:
  // PageInfo::chunk_row (the chunk-relative row index) for all pages in the pass. The start_row
  // field in ColumnChunkDesc is the absolute row index for the whole file. chunk_row in PageInfo is
  // relative to the beginning of the chunk. so in the kernels, chunk.start_row + page.chunk_row
  // gives us the absolute row index
  // Note: chunk_row is already computed if we have column indexes
  if (not _has_page_index) {
    auto key_input  = thrust::make_transform_iterator(pass.pages.d_begin(), get_page_chunk_idx{});
    auto page_input = thrust::make_transform_iterator(pass.pages.d_begin(), get_page_num_rows{});
    thrust::exclusive_scan_by_key(rmm::exec_policy_nosync(_stream),
                                  key_input,
                                  key_input + pass.pages.size(),
                                  page_input,
                                  chunk_row_output_iter{pass.pages.device_ptr()});
  }

  // to compensate for the list row size estimates, force the row count on the last page for each
  // column chunk (each rowgroup) such that it ends on the real known row count. this is so that as
  // we march through the subpasses, we will find that every column cleanly ends up the expected row
  // count at the row group boundary and our split computations work correctly.
  auto iter = thrust::make_counting_iterator(0);
  thrust::for_each(rmm::exec_policy_nosync(_stream),
                   iter,
                   iter + pass.pages.size(),
                   set_final_row_count{pass.pages, pass.chunks});

  pass.chunks.device_to_host_async(_stream);
  pass.pages.device_to_host_async(_stream);
  _stream.synchronize();
}

void reader::impl::preprocess_subpass_pages(bool uses_custom_row_bounds, size_t chunk_read_limit)
{
  auto& pass    = *_pass_itm_data;
  auto& subpass = *pass.subpass;

  // iterate over all input columns and determine if they contain lists.
  // TODO: we could do this once at the file level instead of every time we get in here. the set of
  // columns we are processing does not change over multiple passes/subpasses/output chunks.
  bool has_lists = false;
  for (size_t idx = 0; idx < _input_columns.size(); idx++) {
    auto const& input_col  = _input_columns[idx];
    size_t const max_depth = input_col.nesting_depth();

    auto* cols = &_output_buffers;
    for (size_t l_idx = 0; l_idx < max_depth; l_idx++) {
      auto& out_buf = (*cols)[input_col.nesting[l_idx]];
      cols          = &out_buf.children;

      // if this has a list parent, we have to get column sizes from the
      // data computed during ComputePageSizes
      if (out_buf.user_data & PARQUET_COLUMN_BUFFER_FLAG_HAS_LIST_PARENT) {
        has_lists = true;
        break;
      }
    }
    if (has_lists) { break; }
  }

  // in some cases we will need to do further preprocessing of pages.
  // - if we have lists, the num_rows field in PageInfo will be incorrect coming out of the file
  // - if we are doing a chunked read, we need to compute the size of all string data
  if (has_lists || chunk_read_limit > 0) {
    // computes:
    // PageNestingInfo::num_rows for each page. the true number of rows (taking repetition into
    // account), not just the number of values. PageNestingInfo::size for each level of nesting, for
    // each page.
    //
    // we will be applying a later "trim" pass if skip_rows/num_rows is being used, which can happen
    // if:
    // - user has passed custom row bounds
    // - we will be doing a chunked read
    ComputePageSizes(subpass.pages,
                     pass.chunks,
                     0,  // 0-max size_t. process all possible rows
                     std::numeric_limits<size_t>::max(),
                     true,                  // compute num_rows
                     chunk_read_limit > 0,  // compute string sizes
                     _pass_itm_data->level_type_size,
                     _stream);
  }

  auto iter = thrust::make_counting_iterator(0);

  // copy our now-correct row counts  back to the base pages stored in the pass.
  // only need to do this if we are not processing the whole pass in one subpass
  if (!subpass.single_subpass) {
    thrust::for_each(rmm::exec_policy_nosync(_stream),
                     iter,
                     iter + subpass.pages.size(),
                     update_pass_num_rows{pass.pages, subpass.pages, subpass.page_src_index});
  }

  // computes:
  // PageInfo::chunk_row (the chunk-relative row index) for all pages in the pass. The start_row
  // field in ColumnChunkDesc is the absolute row index for the whole file. chunk_row in PageInfo is
  // relative to the beginning of the chunk. so in the kernels, chunk.start_row + page.chunk_row
  // gives us the absolute row index
  auto key_input  = thrust::make_transform_iterator(pass.pages.d_begin(), get_page_chunk_idx{});
  auto page_input = thrust::make_transform_iterator(pass.pages.d_begin(), get_page_num_rows{});
  thrust::exclusive_scan_by_key(rmm::exec_policy_nosync(_stream),
                                key_input,
                                key_input + pass.pages.size(),
                                page_input,
                                chunk_row_output_iter{pass.pages.device_ptr()});

  // copy chunk row into the subpass pages
  // only need to do this if we are not processing the whole pass in one subpass
  if (!subpass.single_subpass) {
    thrust::for_each(rmm::exec_policy_nosync(_stream),
                     iter,
                     iter + subpass.pages.size(),
                     update_subpass_chunk_row{pass.pages, subpass.pages, subpass.page_src_index});
  }

  // retrieve pages back
  pass.pages.device_to_host_async(_stream);
  if (!subpass.single_subpass) { subpass.pages.device_to_host_async(_stream); }
  _stream.synchronize();

  // at this point we have an accurate row count so we can compute how many rows we will actually be
  // able to decode for this pass. we will have selected a set of pages for each column in the
  // row group, but not every page will have the same number of rows. so, we can only read as many
  // rows as the smallest batch (by column) we have decompressed.
  size_t page_index = 0;
  size_t max_row    = std::numeric_limits<size_t>::max();
  auto const last_pass_row =
    _file_itm_data.input_pass_start_row_count[_file_itm_data._current_input_pass + 1];
  for (size_t idx = 0; idx < subpass.column_page_count.size(); idx++) {
    auto const& last_page = subpass.pages[page_index + (subpass.column_page_count[idx] - 1)];
    auto const& chunk     = pass.chunks[last_page.chunk_idx];

    size_t max_col_row =
      static_cast<size_t>(chunk.start_row + last_page.chunk_row + last_page.num_rows);
    // special case.  list rows can span page boundaries, but we can't tell if that is happening
    // here because we have not yet decoded the pages. the very last row starting in the page may
    // not terminate in the page. to handle this, only decode up to the second to last row in the
    // subpass since we know that will safely completed.
    bool const is_list = chunk.max_level[level_type::REPETITION] > 0;
    if (is_list && max_col_row < last_pass_row) {
      size_t const min_col_row = static_cast<size_t>(chunk.start_row + last_page.chunk_row);
      CUDF_EXPECTS((max_col_row - min_col_row) > 1, "Unexpected short subpass");
      max_col_row--;
    }

    max_row = min(max_row, max_col_row);

    page_index += subpass.column_page_count[idx];
  }
  subpass.skip_rows   = pass.skip_rows + pass.processed_rows;
  auto const pass_end = pass.skip_rows + pass.num_rows;
  max_row             = min(max_row, pass_end);
  subpass.num_rows    = max_row - subpass.skip_rows;

  // now split up the output into chunks as necessary
  compute_output_chunks_for_subpass();
}

void reader::impl::allocate_columns(size_t skip_rows, size_t num_rows, bool uses_custom_row_bounds)
{
  auto& pass    = *_pass_itm_data;
  auto& subpass = *pass.subpass;

  // Should not reach here if there is no page data.
  CUDF_EXPECTS(subpass.pages.size() > 0, "There are no pages present in the subpass");

  // computes:
  // PageNestingInfo::batch_size for each level of nesting, for each page, taking row bounds into
  // account. PageInfo::skipped_values, which tells us where to start decoding in the input to
  // respect the user bounds. It is only necessary to do this second pass if uses_custom_row_bounds
  // is set (if the user has specified artificial bounds).
  if (uses_custom_row_bounds) {
    ComputePageSizes(subpass.pages,
                     pass.chunks,
                     skip_rows,
                     num_rows,
                     false,  // num_rows is already computed
                     false,  // no need to compute string sizes
                     pass.level_type_size,
                     _stream);

    // print_pages(pages, _stream);
  }

  // iterate over all input columns and allocate any associated output
  // buffers if they are not part of a list hierarchy. mark down
  // if we have any list columns that need further processing.
  bool has_lists = false;
  for (size_t idx = 0; idx < _input_columns.size(); idx++) {
    auto const& input_col  = _input_columns[idx];
    size_t const max_depth = input_col.nesting_depth();

    auto* cols = &_output_buffers;
    for (size_t l_idx = 0; l_idx < max_depth; l_idx++) {
      auto& out_buf = (*cols)[input_col.nesting[l_idx]];
      cols          = &out_buf.children;

      // if this has a list parent, we have to get column sizes from the
      // data computed during ComputePageSizes
      if (out_buf.user_data & PARQUET_COLUMN_BUFFER_FLAG_HAS_LIST_PARENT) {
        has_lists = true;
      }
      // if we haven't already processed this column because it is part of a struct hierarchy
      else if (out_buf.size == 0) {
        // add 1 for the offset if this is a list column
        // we're going to start null mask as all valid and then turn bits off if necessary
        out_buf.create_with_mask(
          out_buf.type.id() == type_id::LIST && l_idx < max_depth ? num_rows + 1 : num_rows,
          cudf::mask_state::ALL_VALID,
          _stream,
          _mr);
      }
    }
  }

  // compute output column sizes by examining the pages of the -input- columns
  if (has_lists) {
    std::vector<input_col_info> h_cols_info;
    h_cols_info.reserve(_input_columns.size());
    std::transform(_input_columns.cbegin(),
                   _input_columns.cend(),
                   std::back_inserter(h_cols_info),
                   [](auto& col) -> input_col_info {
                     return {col.schema_idx, static_cast<size_type>(col.nesting_depth())};
                   });

    auto const max_depth =
      (*std::max_element(h_cols_info.cbegin(),
                         h_cols_info.cend(),
                         [](auto& l, auto& r) { return l.nesting_depth < r.nesting_depth; }))
        .nesting_depth;

    auto const d_cols_info = cudf::detail::make_device_uvector_async(
      h_cols_info, _stream, rmm::mr::get_current_device_resource());

    auto const num_keys = _input_columns.size() * max_depth * subpass.pages.size();
    // size iterator. indexes pages by sorted order
    rmm::device_uvector<size_type> size_input{num_keys, _stream};
    thrust::transform(
      rmm::exec_policy(_stream),
      thrust::make_counting_iterator<size_type>(0),
      thrust::make_counting_iterator<size_type>(num_keys),
      size_input.begin(),
      get_page_nesting_size{
        d_cols_info.data(), max_depth, subpass.pages.size(), subpass.pages.device_begin()});
    auto const reduction_keys =
      cudf::detail::make_counting_transform_iterator(0, get_reduction_key{subpass.pages.size()});
    cudf::detail::hostdevice_vector<size_t> sizes{_input_columns.size() * max_depth, _stream};

    // find the size of each column
    thrust::reduce_by_key(rmm::exec_policy(_stream),
                          reduction_keys,
                          reduction_keys + num_keys,
                          size_input.cbegin(),
                          thrust::make_discard_iterator(),
                          sizes.d_begin());

    // for nested hierarchies, compute per-page start offset
    thrust::exclusive_scan_by_key(
      rmm::exec_policy(_stream),
      reduction_keys,
      reduction_keys + num_keys,
      size_input.cbegin(),
      start_offset_output_iterator{
        subpass.pages.device_begin(), 0, d_cols_info.data(), max_depth, subpass.pages.size()});

    sizes.device_to_host_sync(_stream);
    for (size_type idx = 0; idx < static_cast<size_type>(_input_columns.size()); idx++) {
      auto const& input_col = _input_columns[idx];
      auto* cols            = &_output_buffers;
      for (size_type l_idx = 0; l_idx < static_cast<size_type>(input_col.nesting_depth());
           l_idx++) {
        auto& out_buf = (*cols)[input_col.nesting[l_idx]];
        cols          = &out_buf.children;
        // if this buffer is part of a list hierarchy, we need to determine it's
        // final size and allocate it here.
        //
        // for struct columns, higher levels of the output columns are shared between input
        // columns. so don't compute any given level more than once.
        if ((out_buf.user_data & PARQUET_COLUMN_BUFFER_FLAG_HAS_LIST_PARENT) && out_buf.size == 0) {
          auto size = sizes[(idx * max_depth) + l_idx];

          // if this is a list column add 1 for non-leaf levels for the terminating offset
          if (out_buf.type.id() == type_id::LIST && l_idx < max_depth) { size++; }

          // allocate
          // we're going to start null mask as all valid and then turn bits off if necessary
          out_buf.create_with_mask(size, cudf::mask_state::ALL_VALID, _stream, _mr);
        }
      }
    }
  }
}

std::vector<size_t> reader::impl::calculate_page_string_offsets()
{
  auto& pass    = *_pass_itm_data;
  auto& subpass = *pass.subpass;

  auto page_keys = make_page_key_iterator(subpass.pages);

  std::vector<size_t> col_sizes(_input_columns.size(), 0L);
  rmm::device_uvector<size_t> d_col_sizes(col_sizes.size(), _stream);

  // use page_index to fetch page string sizes in the proper order
  auto val_iter = thrust::make_transform_iterator(subpass.pages.device_begin(),
                                                  page_to_string_size{pass.chunks.d_begin()});

  // do scan by key to calculate string offsets for each page
  thrust::exclusive_scan_by_key(rmm::exec_policy_nosync(_stream),
                                page_keys,
                                page_keys + subpass.pages.size(),
                                val_iter,
                                page_offset_output_iter{subpass.pages.device_ptr()});

  // now sum up page sizes
  rmm::device_uvector<int> reduce_keys(col_sizes.size(), _stream);
  thrust::reduce_by_key(rmm::exec_policy_nosync(_stream),
                        page_keys,
                        page_keys + subpass.pages.size(),
                        val_iter,
                        reduce_keys.begin(),
                        d_col_sizes.begin());

  hipMemcpyAsync(col_sizes.data(),
                  d_col_sizes.data(),
                  sizeof(size_t) * col_sizes.size(),
                  hipMemcpyDeviceToHost,
                  _stream);
  _stream.synchronize();

  return col_sizes;
}

}  // namespace cudf::io::parquet::detail
