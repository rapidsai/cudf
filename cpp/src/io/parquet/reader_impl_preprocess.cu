#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2022, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "reader_impl.hpp"
#include "reader_impl_helpers.cuh"

#include <io/utilities/time_utils.cuh>

#include <cudf/detail/iterator.cuh>
#include <cudf/detail/utilities/integer_utils.hpp>
#include <cudf/detail/utilities/vector_factories.hpp>

#include <rmm/cuda_stream_view.hpp>
#include <rmm/exec_policy.hpp>

#include <thrust/binary_search.h>
#include <thrust/functional.h>
#include <thrust/iterator/constant_iterator.h>
#include <thrust/iterator/discard_iterator.h>
#include <thrust/iterator/iterator_categories.h>
#include <thrust/iterator/transform_iterator.h>
#include <thrust/reduce.h>
#include <thrust/scan.h>
#include <thrust/sort.h>
#include <thrust/transform.h>

namespace cudf::io::detail::parquet {

namespace {

#if defined(PREPROCESS_DEBUG)
void print_pages(hostdevice_vector<gpu::PageInfo>& pages, rmm::cuda_stream_view _stream)
{
  pages.device_to_host(_stream, true);
  for (size_t idx = 0; idx < pages.size(); idx++) {
    auto const& p = pages[idx];
    // skip dictionary pages
    if (p.flags & gpu::PAGEINFO_FLAGS_DICTIONARY) { continue; }
    printf(
      "P(%lu, s:%d): chunk_row(%d), num_rows(%d), skipped_values(%d), skipped_leaf_values(%d)\n",
      idx,
      p.src_col_schema,
      p.chunk_row,
      p.num_rows,
      p.skipped_values,
      p.skipped_leaf_values);
  }
}

void print_chunks(hostdevice_vector<gpu::ColumnChunkDesc>& chunks, rmm::cuda_stream_view _stream)
{
  chunks.device_to_host(_stream, true);
  for (size_t idx = 0; idx < chunks.size(); idx++) {
    auto const& c = chunks[idx];
    printf("C(%lu, s:%d): num_values(%lu), start_row(%lu), num_rows(%u)\n",
           idx,
           c.src_col_schema,
           c.num_values,
           c.start_row,
           c.num_rows);
  }
}
#endif  // PREPROCESS_DEBUG

struct cumulative_row_info {
  size_t row_count;   // cumulative row count
  size_t size_bytes;  // cumulative size in bytes
  int key;            // schema index
};
struct cumulative_row_sum {
  cumulative_row_info operator()
    __device__(cumulative_row_info const& a, cumulative_row_info const& b) const
  {
    return cumulative_row_info{a.row_count + b.row_count, a.size_bytes + b.size_bytes, a.key};
  }
};

struct row_size_functor {
  __device__ size_t validity_size(size_t num_rows, bool nullable)
  {
    return nullable ? (cudf::util::div_rounding_up_safe(num_rows, size_t{32}) / 8) : 0;
  }

  template <typename T>
  __device__ size_t operator()(size_t num_rows, bool nullable)
  {
    auto const element_size = sizeof(device_storage_type_t<T>);
    return (element_size * num_rows) + validity_size(num_rows, nullable);
  }
};

template <>
__device__ size_t row_size_functor::operator()<list_view>(size_t num_rows, bool nullable)
{
  auto const offset_size = sizeof(offset_type);
  return (offset_size * (num_rows + 1)) + validity_size(num_rows, nullable);
}

template <>
__device__ size_t row_size_functor::operator()<struct_view>(size_t num_rows, bool nullable)
{
  return validity_size(num_rows, nullable);
}

template <>
__device__ size_t row_size_functor::operator()<string_view>(size_t num_rows, bool nullable)
{
  // only returns the size of offsets and validity. the size of the actual string chars
  // is tracked separately.
  auto const offset_size = sizeof(offset_type);
  return (offset_size * (num_rows + 1)) + validity_size(num_rows, nullable);
}

struct get_cumulative_row_info {
  gpu::PageInfo const* const pages;

  cumulative_row_info operator() __device__(size_type index)
  {
    auto const& page = pages[index];
    if (page.flags & gpu::PAGEINFO_FLAGS_DICTIONARY) {
      return cumulative_row_info{0, 0, page.src_col_schema};
    }
    size_t const row_count = page.nesting[0].size;
    return cumulative_row_info{
      row_count,
      // note: the size of the actual char bytes for strings is tracked in the `str_bytes` field, so
      // the row_size_functor{} itself is only returning the size of offsets+validity
      cudf::type_dispatcher(data_type{page.type}, row_size_functor{}, row_count, false) +
        page.str_bytes,
      page.src_col_schema};
  }
};

struct row_total_size {
  cumulative_row_info const* const c_info;
  size_type const* const key_offsets;
  size_t const num_keys;

  __device__ cumulative_row_info operator()(cumulative_row_info const& i)
  {
    // sum sizes for each input column at this row
    size_t sum = 0;
    for (int idx = 0; idx < num_keys; idx++) {
      auto const start = key_offsets[idx];
      auto const end   = key_offsets[idx + 1];
      auto iter        = cudf::detail::make_counting_transform_iterator(
        0, [&] __device__(size_type i) { return c_info[start + i].row_count; });
      auto const page_index =
        (thrust::lower_bound(thrust::seq, iter, iter + (end - start), i.row_count) - iter) + start;
      // printf("KI(%d): start(%d), end(%d), page_index(%d), size_bytes(%lu)\n", idx, start, end,
      // (int)page_index, c_info[page_index].size_bytes);
      sum += c_info[page_index].size_bytes;
    }
    return {i.row_count, sum};
  }
};

std::vector<gpu::chunk_read_info> find_splits(std::vector<cumulative_row_info> const& sizes,
                                              size_type num_rows,
                                              size_t chunked_read_size)
{
  // now we have an array of {row_count, real output bytes}. just walk through it and generate
  // splits.
  // TODO: come up with a clever way to do this entirely in parallel. For now, as long as batch
  // sizes are reasonably large, this shouldn't iterate too many times
  std::vector<gpu::chunk_read_info> splits;
  {
    size_t cur_pos             = 0;
    size_t cur_cumulative_size = 0;
    size_t cur_row_count       = 0;
    auto start = thrust::make_transform_iterator(sizes.begin(), [&](cumulative_row_info const& i) {
      return i.size_bytes - cur_cumulative_size;
    });
    auto end   = start + sizes.size();
    while (cur_row_count < static_cast<size_t>(num_rows)) {
      int64_t p = thrust::lower_bound(
                    thrust::seq, start + cur_pos, end, static_cast<size_t>(chunked_read_size)) -
                  start;

      // if we're past the end, or if the returned bucket is > than the chunked_read_size, move back
      // one.
      if (static_cast<size_t>(p) >= sizes.size() ||
          (sizes[p].size_bytes - cur_cumulative_size > static_cast<size_t>(chunked_read_size))) {
        p--;
      }

      // best-try. if we can't find something that'll fit, we have to go bigger. we're doing this in
      // a loop because all of the cumulative sizes for all the pages are sorted into one big list.
      // so if we had two columns, both of which had an entry {1000, 10000}, that entry would be in
      // the list twice. so we have to iterate until we skip past all of them.  The idea is that we
      // either do this, or we have to call unique() on the input first.
      while (p < (static_cast<int64_t>(sizes.size()) - 1) &&
             (sizes[p].row_count == cur_row_count || p < 0)) {
        p++;
      }

      auto const start_row = cur_row_count;
      cur_row_count        = sizes[p].row_count;
      splits.push_back(gpu::chunk_read_info{start_row, cur_row_count - start_row});
      // printf("Split: {%lu, %lu}\n", splits.back().skip_rows, splits.back().num_rows);
      cur_pos             = p;
      cur_cumulative_size = sizes[p].size_bytes;
    }
  }
  return splits;
}

std::vector<gpu::chunk_read_info> compute_splits(hostdevice_vector<gpu::PageInfo>& pages,
                                                 gpu::chunk_intermediate_data const& id,
                                                 size_type num_rows,
                                                 size_type chunked_read_size,
                                                 rmm::cuda_stream_view stream)
{
  auto const& page_keys  = id.page_keys;
  auto const& page_index = id.page_index;

  // generate cumulative row counts and sizes
  rmm::device_uvector<cumulative_row_info> c_info(page_keys.size(), stream);
  // convert PageInfo to cumulative_row_info
  auto page_input = thrust::make_transform_iterator(page_index.begin(),
                                                    get_cumulative_row_info{pages.device_ptr()});
  thrust::inclusive_scan_by_key(rmm::exec_policy(stream),
                                page_keys.begin(),
                                page_keys.end(),
                                page_input,
                                c_info.begin(),
                                thrust::equal_to{},
                                cumulative_row_sum{});
  // clang-format off
  /*
  stream.synchronize();
  pages.device_to_host(stream);
  std::vector<int> h_page_index(pages.size());
  hipMemcpyAsync(h_page_index.data(), page_index.begin(), sizeof(int) * pages.size(), hipMemcpyDeviceToHost, stream);
  stream.synchronize();
  for(size_t idx=0; idx<pages.size(); idx++){
    auto const& page = pages[h_page_index[idx]];
    if(page.flags & gpu::PAGEINFO_FLAGS_DICTIONARY){
      continue;
    }
    gpu::PageNestingInfo pni;
    hipMemcpy(&pni, &page.nesting[0], sizeof(gpu::PageNestingInfo), hipMemcpyDeviceToHost);
    printf("P(%lu): schema(%d), num_rows(%d), nesting size 0(%d), str_bytes(%d)\n", idx, page.src_col_schema, page.num_rows, pni.size, page.str_bytes);
  }
  printf("---------\n");
  std::vector<cumulative_row_info> h_c_info(page_keys.size());
  hipMemcpy(h_c_info.data(), c_info.data(), sizeof(cumulative_row_info) * page_keys.size(), hipMemcpyDeviceToHost);
  for(size_t idx=0; idx<page_keys.size(); idx++){
    printf("C(%lu): %lu, %lu\n", idx, h_c_info[idx].row_count, h_c_info[idx].size_bytes);
  }
  */
  // clang-format on

  // sort by row count
  rmm::device_uvector<cumulative_row_info> c_info_sorted{c_info, stream};
  thrust::sort(rmm::exec_policy(stream),
               c_info_sorted.begin(),
               c_info_sorted.end(),
               [] __device__(cumulative_row_info const& a, cumulative_row_info const& b) {
                 return a.row_count < b.row_count;
               });

  // generate key offsets (offsets to the start of each partition of keys). worst case is 1 page per
  // key
  rmm::device_uvector<size_type> key_offsets(page_keys.size() + 1, stream);
  auto [_, key_offsets_end]    = thrust::reduce_by_key(rmm::exec_policy(stream),
                                                    page_keys.begin(),
                                                    page_keys.end(),
                                                    thrust::make_constant_iterator(1),
                                                    thrust::make_discard_iterator(),
                                                    key_offsets.begin());
  size_t const num_unique_keys = key_offsets_end - key_offsets.begin();
  // clang-format off
  /*
  stream.synchronize();
  printf("Num keys: %d\n", (int)num_unique_keys);
  std::vector<size_type> h_key_offsets(num_unique_keys);
  hipMemcpy(h_key_offsets.data(), key_offsets.data(), sizeof(size_type) * num_unique_keys, hipMemcpyDeviceToHost);
  for(size_t idx=0; idx<num_unique_keys; idx++){
    printf("Offset sizes(%lu): %d\n", idx, h_key_offsets[idx]);
  }
  */
  // clang-format on

  thrust::exclusive_scan(
    rmm::exec_policy(stream), key_offsets.begin(), key_offsets.end(), key_offsets.begin());
  // clang-format off
  /*
  stream.synchronize();
  hipMemcpy(h_key_offsets.data(), key_offsets.data(), sizeof(size_type) * (num_unique_keys + 1), hipMemcpyDeviceToHost);
  for(size_t idx=0; idx<num_unique_keys+1; idx++){
    printf("Offset values(%lu): %d\n", idx, h_key_offsets[idx]);
  }
  */
  // clang-format on

  // adjust the cumulative info such that for each row count, the size includes any pages that span
  // that row count. this is so that if we have this case:
  //              page row counts
  // Column A:    0 <----> 100 <----> 200
  // Column B:    0 <---------------> 200 <--------> 400
  //                        |
  // if we decide to split at row 100, we don't really know the actual amount of bytes in column B
  // at that point.  So we have to proceed as if we are taking the bytes from all 200 rows of that
  // page.
  //
  rmm::device_uvector<cumulative_row_info> adjusted(c_info.size(), stream);
  thrust::transform(rmm::exec_policy(stream),
                    c_info_sorted.begin(),
                    c_info_sorted.end(),
                    adjusted.begin(),
                    row_total_size{c_info.data(), key_offsets.data(), num_unique_keys});

  // bring back to the cpu
  std::vector<cumulative_row_info> h_adjusted(adjusted.size());
  hipMemcpyAsync(h_adjusted.data(),
                  adjusted.data(),
                  sizeof(cumulative_row_info) * c_info.size(),
                  hipMemcpyDeviceToHost,
                  stream);
  stream.synchronize();
  // clang-format off
  /*
  for(size_t idx=0; idx<h_adjusted.size(); idx++){
    printf("A(%lu): %lu, %lu\n", idx, h_adjusted[idx].row_count, h_adjusted[idx].size_bytes);
  }
  */
  // clang-format on

  return find_splits(h_adjusted, num_rows, chunked_read_size);
}

struct get_page_chunk_idx {
  __device__ size_type operator()(gpu::PageInfo const& page) { return page.chunk_idx; }
};

struct get_page_num_rows {
  __device__ size_type operator()(gpu::PageInfo const& page) { return page.num_rows; }
};

struct get_page_schema {
  __device__ size_type operator()(gpu::PageInfo const& page) { return page.src_col_schema; }
};

struct get_page_nesting_size {
  size_type const src_col_schema;
  size_type const depth;
  gpu::PageInfo const* const pages;

  __device__ size_type operator()(int index)
  {
    auto const& page = pages[index];
    if (page.src_col_schema != src_col_schema || page.flags & gpu::PAGEINFO_FLAGS_DICTIONARY) {
      return 0;
    }
    return page.nesting[depth].batch_size;
  }
};

struct chunk_row_output_iter {
  gpu::PageInfo* p;
  using value_type        = size_type;
  using difference_type   = size_type;
  using pointer           = size_type*;
  using reference         = size_type&;
  using iterator_category = thrust::output_device_iterator_tag;

  __host__ __device__ chunk_row_output_iter operator+(int i)
  {
    return chunk_row_output_iter{p + i};
  }

  __host__ __device__ void operator++() { p++; }

  __device__ reference operator[](int i) { return p[i].chunk_row; }
  __device__ reference operator*() { return p->chunk_row; }
  // __device__ void operator=(value_type v) { p->chunk_row = v; }
};

struct start_offset_output_iterator {
  gpu::PageInfo* pages;
  int const* page_indices;
  int cur_index;
  int src_col_schema;
  int nesting_depth;
  int empty               = 0;
  using value_type        = size_type;
  using difference_type   = size_type;
  using pointer           = size_type*;
  using reference         = size_type&;
  using iterator_category = thrust::output_device_iterator_tag;

  __host__ __device__ void operator=(start_offset_output_iterator const& other)
  {
    pages          = other.pages;
    page_indices   = other.page_indices;
    cur_index      = other.cur_index;
    src_col_schema = other.src_col_schema;
    nesting_depth  = other.nesting_depth;
  }

  __host__ __device__ start_offset_output_iterator operator+(int i)
  {
    return start_offset_output_iterator{
      pages, page_indices, cur_index + i, src_col_schema, nesting_depth};
  }

  __host__ __device__ void operator++() { cur_index++; }

  __device__ reference operator[](int i) { return dereference(cur_index + i); }
  __device__ reference operator*() { return dereference(cur_index); }

 private:
  __device__ reference dereference(int index)
  {
    gpu::PageInfo const& p = pages[page_indices[index]];
    if (p.src_col_schema != src_col_schema || p.flags & gpu::PAGEINFO_FLAGS_DICTIONARY) {
      return empty;
    }
    return p.nesting[nesting_depth].page_start_value;
  }
};

}  // anonymous namespace

void reader::impl::preprocess_columns(hostdevice_vector<gpu::ColumnChunkDesc>& chunks,
                                      hostdevice_vector<gpu::PageInfo>& pages,
                                      size_t min_row,
                                      size_t num_rows,
                                      bool uses_custom_row_bounds,
                                      size_type chunked_read_size)
{
  // iterate over all input columns and determine if they contain lists so we can further
  // preprocess them.
  bool has_lists = false;
  for (size_t idx = 0; idx < _input_columns.size(); idx++) {
    auto const& input_col  = _input_columns[idx];
    size_t const max_depth = input_col.nesting_depth();

    auto* cols = &_output_columns;
    for (size_t l_idx = 0; l_idx < max_depth; l_idx++) {
      auto& out_buf = (*cols)[input_col.nesting[l_idx]];
      cols          = &out_buf.children;

      // if this has a list parent, we will have to do further work in gpu::PreprocessColumnData
      // to know how big this buffer actually is.
      if (out_buf.user_data & PARQUET_COLUMN_BUFFER_FLAG_HAS_LIST_PARENT) {
        has_lists = true;
        break;
      }
    }
    if (has_lists) { break; }
  }

  // generate string dict indices if necessary
  {
    auto is_dict_chunk = [](const gpu::ColumnChunkDesc& chunk) {
      return (chunk.data_type & 0x7) == BYTE_ARRAY && chunk.num_dict_pages > 0;
    };

    // Count the number of string dictionary entries
    // NOTE: Assumes first page in the chunk is always the dictionary page
    size_t total_str_dict_indexes = 0;
    for (size_t c = 0, page_count = 0; c < chunks.size(); c++) {
      if (is_dict_chunk(chunks[c])) {
        total_str_dict_indexes += pages[page_count].num_input_values;
      }
      page_count += chunks[c].max_num_pages;
    }

    // Build index for string dictionaries since they can't be indexed
    // directly due to variable-sized elements
    _chunk_itm_data.str_dict_index =
      cudf::detail::make_zeroed_device_uvector_async<string_index_pair>(total_str_dict_indexes,
                                                                        _stream);

    // Update chunks with pointers to string dict indices
    for (size_t c = 0, page_count = 0, str_ofs = 0; c < chunks.size(); c++) {
      input_column_info const& input_col = _input_columns[chunks[c].src_col_index];
      CUDF_EXPECTS(input_col.schema_idx == chunks[c].src_col_schema,
                   "Column/page schema index mismatch");
      if (is_dict_chunk(chunks[c])) {
        chunks[c].str_dict_index = _chunk_itm_data.str_dict_index.data() + str_ofs;
        str_ofs += pages[page_count].num_input_values;
      }

      // column_data_base will always point to leaf data, even for nested types.
      page_count += chunks[c].max_num_pages;
    }

    if (total_str_dict_indexes > 0) {
      chunks.host_to_device(_stream);
      gpu::BuildStringDictionaryIndex(chunks.device_ptr(), chunks.size(), _stream);
    }
  }

  // intermediate data we will need for further chunked reads
  if (has_lists || chunked_read_size > 0) {
    // computes:
    // PageNestingInfo::num_rows for each page. the true number of rows (taking repetition into
    // account), not just the number of values. PageNestingInfo::size for each level of nesting, for
    // each page.
    //
    // we will be applying a later "trim" pass if skip_rows/num_rows is being used, which can happen
    // if:
    // - user has passed custom row bounds
    // - if we will be doing a chunked read
    gpu::ComputePageSizes(pages,
                          chunks,
                          0,  // 0-max size_t. process all possible rows
                          std::numeric_limits<size_t>::max(),
                          true,                   // compute num_rows
                          chunked_read_size > 0,  // compute string sizes
                          _stream);

    // computes:
    // PageInfo::chunk_row (the absolute start row index) for all pages
    // Note: this is doing some redundant work for pages in flat hierarchies.  chunk_row has already
    // been computed during header decoding. the overall amount of work here is very small though.
    auto key_input  = thrust::make_transform_iterator(pages.device_ptr(), get_page_chunk_idx{});
    auto page_input = thrust::make_transform_iterator(pages.device_ptr(), get_page_num_rows{});
    thrust::exclusive_scan_by_key(rmm::exec_policy(_stream),
                                  key_input,
                                  key_input + pages.size(),
                                  page_input,
                                  chunk_row_output_iter{pages.device_ptr()});

    // compute page ordering.
    //
    // ordering of pages is by input column schema, repeated across row groups.  so
    // if we had 3 columns, each with 2 pages, and 1 row group, our schema values might look like
    //
    // 1, 1, 2, 2, 3, 3
    //
    // However, if we had more than one row group, the pattern would be
    //
    // 1, 1, 2, 2, 3, 3, 1, 1, 2, 2, 3, 3
    // ^ row group 0     |
    //                   ^ row group 1
    //
    // To use exclusive_scan_by_key, the ordering we actually want is
    //
    // 1, 1, 1, 1, 2, 2, 2, 2, 3, 3, 3, 3
    //
    // We also need to preserve key-relative page ordering, so we need to use a stable sort.
    _chunk_itm_data.page_keys  = rmm::device_uvector<int>(pages.size(), _stream);
    _chunk_itm_data.page_index = rmm::device_uvector<int>(pages.size(), _stream);
    auto& page_keys            = _chunk_itm_data.page_keys;
    auto& page_index           = _chunk_itm_data.page_index;
    {
      thrust::transform(rmm::exec_policy(_stream),
                        pages.device_ptr(),
                        pages.device_ptr() + pages.size(),
                        page_keys.begin(),
                        get_page_schema{});

      thrust::sequence(rmm::exec_policy(_stream), page_index.begin(), page_index.end());
      thrust::stable_sort_by_key(rmm::exec_policy(_stream),
                                 page_keys.begin(),
                                 page_keys.end(),
                                 page_index.begin(),
                                 thrust::less<int>());
    }

    // retrieve pages back
    pages.device_to_host(_stream, true);

    // print_pages(pages, _stream);
  }

  // compute splits if necessary.
  _chunk_read_info =
    chunked_read_size > 0
      ? compute_splits(pages, _chunk_itm_data, num_rows, chunked_read_size, _stream)
      : std::vector<gpu::chunk_read_info>{{min_row, num_rows}};
}

void reader::impl::allocate_columns(hostdevice_vector<gpu::ColumnChunkDesc>& chunks,
                                    hostdevice_vector<gpu::PageInfo>& pages,
                                    gpu::chunk_intermediate_data const& id,
                                    size_t min_row,
                                    size_t num_rows,
                                    bool uses_custom_row_bounds)
{
  // computes:
  // PageNestingInfo::batch_size for each level of nesting, for each page, taking row bounds into
  // account. PageInfo::skipped_values, which tells us where to start decoding in the input to
  // respect the user bounds. It is only necessary to do this second pass if uses_custom_row_bounds
  // is set (if the user has specified artifical bounds).
  if (uses_custom_row_bounds) {
    gpu::ComputePageSizes(pages,
                          chunks,
                          min_row,
                          num_rows,
                          false,  // num_rows is already computed
                          false,  // no need to compute string sizes
                          _stream);
    // print_pages(pages, _stream);
  }

  // iterate over all input columns and allocate any associated output
  // buffers if they are not part of a list hierarchy. mark down
  // if we have any list columns that need further processing.
  bool has_lists = false;
  for (size_t idx = 0; idx < _input_columns.size(); idx++) {
    auto const& input_col  = _input_columns[idx];
    size_t const max_depth = input_col.nesting_depth();

    auto* cols = &_output_columns;
    for (size_t l_idx = 0; l_idx < max_depth; l_idx++) {
      auto& out_buf = (*cols)[input_col.nesting[l_idx]];
      cols          = &out_buf.children;

      // if this has a list parent, we will have to do further work in gpu::PreprocessColumnData
      // to know how big this buffer actually is.
      if (out_buf.user_data & PARQUET_COLUMN_BUFFER_FLAG_HAS_LIST_PARENT) {
        has_lists = true;
      }
      // if we haven't already processed this column because it is part of a struct hierarchy
      else if (out_buf.size == 0) {
        // add 1 for the offset if this is a list column
        out_buf.create(
          out_buf.type.id() == type_id::LIST && l_idx < max_depth ? num_rows + 1 : num_rows,
          _stream,
          _mr);
      }
    }
  }

  // compute output column sizes by examining the pages of the -input- columns
  if (has_lists) {
    auto& page_keys  = _chunk_itm_data.page_keys;
    auto& page_index = _chunk_itm_data.page_index;
    for (size_t idx = 0; idx < _input_columns.size(); idx++) {
      auto const& input_col = _input_columns[idx];
      auto src_col_schema   = input_col.schema_idx;
      size_t max_depth      = input_col.nesting_depth();

      auto* cols = &_output_columns;
      for (size_t l_idx = 0; l_idx < input_col.nesting_depth(); l_idx++) {
        auto& out_buf = (*cols)[input_col.nesting[l_idx]];
        cols          = &out_buf.children;

        // size iterator. indexes pages by sorted order
        auto size_input = thrust::make_transform_iterator(
          page_index.begin(),
          get_page_nesting_size{src_col_schema, static_cast<size_type>(l_idx), pages.device_ptr()});

        // if this buffer is part of a list hierarchy, we need to determine it's
        // final size and allocate it here.
        //
        // for struct columns, higher levels of the output columns are shared between input
        // columns. so don't compute any given level more than once.
        if ((out_buf.user_data & PARQUET_COLUMN_BUFFER_FLAG_HAS_LIST_PARENT) && out_buf.size == 0) {
          int size =
            thrust::reduce(rmm::exec_policy(_stream), size_input, size_input + pages.size());

          // if this is a list column add 1 for non-leaf levels for the terminating offset
          if (out_buf.type.id() == type_id::LIST && l_idx < max_depth) { size++; }

          // allocate
          out_buf.create(size, _stream, _mr);
        }

        // for nested hierarchies, compute per-page start offset
        if (input_col.has_repetition) {
          thrust::exclusive_scan_by_key(
            rmm::exec_policy(_stream),
            page_keys.begin(),
            page_keys.end(),
            size_input,
            start_offset_output_iterator{pages.device_ptr(),
                                         page_index.begin(),
                                         0,
                                         static_cast<int>(src_col_schema),
                                         static_cast<int>(l_idx)});
        }
      }
    }
  }
}

namespace {
/**
 * @brief Function that returns the required the number of bits to store a value
 */
template <typename T = uint8_t>
T required_bits(uint32_t max_level)
{
  return static_cast<T>(CompactProtocolReader::NumRequiredBits(max_level));
}

/**
 * @brief Converts cuDF units to Parquet units.
 *
 * @return A tuple of Parquet type width, Parquet clock rate and Parquet decimal type.
 */
std::tuple<int32_t, int32_t, int8_t> conversion_info(type_id column_type_id,
                                                     type_id timestamp_type_id,
                                                     parquet::Type physical,
                                                     int8_t converted,
                                                     int32_t length)
{
  int32_t type_width = (physical == parquet::FIXED_LEN_BYTE_ARRAY) ? length : 0;
  int32_t clock_rate = 0;
  if (column_type_id == type_id::INT8 or column_type_id == type_id::UINT8) {
    type_width = 1;  // I32 -> I8
  } else if (column_type_id == type_id::INT16 or column_type_id == type_id::UINT16) {
    type_width = 2;  // I32 -> I16
  } else if (column_type_id == type_id::INT32) {
    type_width = 4;  // str -> hash32
  } else if (is_chrono(data_type{column_type_id})) {
    clock_rate = to_clockrate(timestamp_type_id);
  }

  int8_t converted_type = converted;
  if (converted_type == parquet::DECIMAL && column_type_id != type_id::FLOAT64 &&
      not cudf::is_fixed_point(data_type{column_type_id})) {
    converted_type = parquet::UNKNOWN;  // Not converting to float64 or decimal
  }
  return std::make_tuple(type_width, clock_rate, converted_type);
}

}  // namespace

std::pair<size_type, size_type> reader::impl::preprocess_file(
  size_type skip_rows,
  size_type num_rows,
  const std::vector<std::vector<size_type>>& row_group_list)
{
  //  printf("\n\n\n\npreprocess========================\n");

  // Select only row groups required
  // Note: `skip_rows` and `num_rows` will be modified in this function.
  const auto selected_row_groups =
    _metadata->select_row_groups(row_group_list, skip_rows, num_rows);

  // TODO: fix this
  if (selected_row_groups.size() == 0 || _input_columns.size() == 0) {
    return {skip_rows, num_rows};
  }

  // TODO: fix this.
  // Need to check if the file actually has data.
  _file_itm_data.has_data = true;

  // Descriptors for all the chunks that make up the selected columns
  const auto num_input_columns = _input_columns.size();
  const auto num_chunks        = selected_row_groups.size() * num_input_columns;
  _file_itm_data.chunks        = hostdevice_vector<gpu::ColumnChunkDesc>(0, num_chunks, _stream);

  // Association between each column chunk and its source
  std::vector<size_type> chunk_source_map(num_chunks);

  // Tracker for eventually deallocating compressed and uncompressed data
  _file_itm_data.raw_page_data = std::vector<std::unique_ptr<datasource::buffer>>(num_chunks);

  // Keep track of column chunk file offsets
  std::vector<size_t> column_chunk_offsets(num_chunks);

  // Initialize column chunk information
  size_t total_decompressed_size = 0;
  auto remaining_rows            = num_rows;
  std::vector<std::future<void>> read_rowgroup_tasks;
  for (const auto& rg : selected_row_groups) {
    const auto& row_group       = _metadata->get_row_group(rg.index, rg.source_index);
    auto const row_group_start  = rg.start_row;
    auto const row_group_source = rg.source_index;
    auto const row_group_rows   = std::min<int>(remaining_rows, row_group.num_rows);
    auto const io_chunk_idx     = _file_itm_data.chunks.size();

    // generate ColumnChunkDesc objects for everything to be decoded (all input columns)
    for (size_t i = 0; i < num_input_columns; ++i) {
      auto col = _input_columns[i];
      // look up metadata
      auto& col_meta = _metadata->get_column_metadata(rg.index, rg.source_index, col.schema_idx);
      auto& schema   = _metadata->get_schema(col.schema_idx);

      auto [type_width, clock_rate, converted_type] =
        conversion_info(to_type_id(schema, _strings_to_categorical, _timestamp_type.id()),
                        _timestamp_type.id(),
                        schema.type,
                        schema.converted_type,
                        schema.type_length);

      column_chunk_offsets[_file_itm_data.chunks.size()] =
        (col_meta.dictionary_page_offset != 0)
          ? std::min(col_meta.data_page_offset, col_meta.dictionary_page_offset)
          : col_meta.data_page_offset;

      _file_itm_data.chunks.push_back(
        gpu::ColumnChunkDesc(col_meta.total_compressed_size,
                             nullptr,
                             col_meta.num_values,
                             schema.type,
                             type_width,
                             row_group_start,
                             row_group_rows,
                             schema.max_definition_level,
                             schema.max_repetition_level,
                             _metadata->get_output_nesting_depth(col.schema_idx),
                             required_bits(schema.max_definition_level),
                             required_bits(schema.max_repetition_level),
                             col_meta.codec,
                             converted_type,
                             schema.logical_type,
                             schema.decimal_scale,
                             clock_rate,
                             i,
                             col.schema_idx));

      // Map each column chunk to its column index and its source index
      chunk_source_map[_file_itm_data.chunks.size() - 1] = row_group_source;

      if (col_meta.codec != Compression::UNCOMPRESSED) {
        total_decompressed_size += col_meta.total_uncompressed_size;
      }
    }
    // Read compressed chunk data to device memory
    read_rowgroup_tasks.push_back(read_column_chunks(_file_itm_data.raw_page_data,
                                                     _file_itm_data.chunks,
                                                     io_chunk_idx,
                                                     _file_itm_data.chunks.size(),
                                                     column_chunk_offsets,
                                                     chunk_source_map));

    remaining_rows -= row_group.num_rows;
  }
  for (auto& task : read_rowgroup_tasks) {
    task.wait();
  }
  assert(remaining_rows <= 0);

  // Process dataset chunk pages into output columns
  const auto total_pages    = count_page_headers(_file_itm_data.chunks);
  _file_itm_data.pages_info = hostdevice_vector<gpu::PageInfo>(total_pages, total_pages, _stream);

  if (total_pages > 0) {
    // decoding of column/page information
    decode_page_headers(_file_itm_data.chunks, _file_itm_data.pages_info);
    if (total_decompressed_size > 0) {
      _file_itm_data.decomp_page_data =
        decompress_page_data(_file_itm_data.chunks, _file_itm_data.pages_info);
      // Free compressed data
      for (size_t c = 0; c < _file_itm_data.chunks.size(); c++) {
        if (_file_itm_data.chunks[c].codec != parquet::Compression::UNCOMPRESSED) {
          _file_itm_data.raw_page_data[c].reset();
          // TODO: Check if this is called
        }
      }
    }

    // build output column info
    // walk the schema, building out_buffers that mirror what our final cudf columns will look
    // like. important : there is not necessarily a 1:1 mapping between input columns and output
    // columns. For example, parquet does not explicitly store a ColumnChunkDesc for struct
    // columns. The "structiness" is simply implied by the schema.  For example, this schema:
    //  required group field_id=1 name {
    //    required binary field_id=2 firstname (String);
    //    required binary field_id=3 middlename (String);
    //    required binary field_id=4 lastname (String);
    // }
    // will only contain 3 columns of data (firstname, middlename, lastname).  But of course
    // "name" is a struct column that we want to return, so we have to make sure that we
    // create it ourselves.
    // std::vector<output_column_info> output_info = build_output_column_info();

    // nesting information (sizes, etc) stored -per page-
    // note : even for flat schemas, we allocate 1 level of "nesting" info

    allocate_nesting_info(
      _file_itm_data.chunks, _file_itm_data.pages_info, _file_itm_data.page_nesting_info);
  }

  return {skip_rows, num_rows};
}

}  // namespace cudf::io::detail::parquet
