#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

/**
 * @file reader_impl.cu
 * @brief cuDF-IO Parquet reader class implementation
 */

#include "reader_impl.hpp"

#include <io/comp/gpuinflate.h>

#include <cudf/table/table.hpp>
#include <cudf/utilities/error.hpp>
#include <cudf/utilities/traits.hpp>

#include <rmm/thrust_rmm_allocator.h>
#include <rmm/device_buffer.hpp>

#include <algorithm>
#include <array>
#include <numeric>
#include <regex>

namespace cudf {
namespace io {
namespace detail {
namespace parquet {
// Import functionality that's independent of legacy code
using namespace cudf::io::parquet;
using namespace cudf::io;

namespace {
/**
 * @brief Function that translates Parquet datatype to cuDF type enum
 */
constexpr type_id to_type_id(parquet::Type physical,
                             parquet::ConvertedType logical,
                             bool strings_to_categorical,
                             type_id timestamp_type_id,
                             int32_t decimal_scale)
{
  // Logical type used for actual data interpretation; the legacy converted type
  // is superceded by 'logical' type whenever available.
  switch (logical) {
    case parquet::UINT_8: return type_id::UINT8;
    case parquet::INT_8: return type_id::INT8;
    case parquet::UINT_16: return type_id::UINT16;
    case parquet::INT_16: return type_id::INT16;
    case parquet::UINT_32: return type_id::UINT32;
    case parquet::UINT_64: return type_id::UINT64;
    case parquet::DATE: return type_id::TIMESTAMP_DAYS;
    case parquet::TIMESTAMP_MICROS:
      return (timestamp_type_id != type_id::EMPTY) ? timestamp_type_id
                                                   : type_id::TIMESTAMP_MICROSECONDS;
    case parquet::TIMESTAMP_MILLIS:
      return (timestamp_type_id != type_id::EMPTY) ? timestamp_type_id
                                                   : type_id::TIMESTAMP_MILLISECONDS;
    case parquet::DECIMAL:
      if (decimal_scale != 0 || (physical != parquet::INT32 && physical != parquet::INT64)) {
        return type_id::FLOAT64;
      }
      break;
    default: break;
  }

  // Physical storage type supported by Parquet; controls the on-disk storage
  // format in combination with the encoding type.
  switch (physical) {
    case parquet::BOOLEAN: return type_id::BOOL8;
    case parquet::INT32: return type_id::INT32;
    case parquet::INT64: return type_id::INT64;
    case parquet::FLOAT: return type_id::FLOAT32;
    case parquet::DOUBLE: return type_id::FLOAT64;
    case parquet::BYTE_ARRAY:
    case parquet::FIXED_LEN_BYTE_ARRAY:
      // Can be mapped to INT32 (32-bit hash) or STRING
      return strings_to_categorical ? type_id::INT32 : type_id::STRING;
    case parquet::INT96:
      return (timestamp_type_id != type_id::EMPTY) ? timestamp_type_id
                                                   : type_id::TIMESTAMP_NANOSECONDS;
    default: break;
  }

  return type_id::EMPTY;
}

/**
 * @brief Function that translates cuDF time unit to Parquet clock frequency
 */
constexpr int32_t to_clockrate(type_id timestamp_type_id)
{
  switch (timestamp_type_id) {
    case type_id::TIMESTAMP_SECONDS: return 1;
    case type_id::TIMESTAMP_MILLISECONDS: return 1000;
    case type_id::TIMESTAMP_MICROSECONDS: return 1000000;
    case type_id::TIMESTAMP_NANOSECONDS: return 1000000000;
    default: return 0;
  }
}

/**
 * @brief Function that returns the required the number of bits to store a value
 */
template <typename T = uint8_t>
T required_bits(uint32_t max_level)
{
  return static_cast<T>(CompactProtocolReader::NumRequiredBits(max_level));
}

std::tuple<int32_t, int32_t, int8_t> conversion_info(type_id column_type_id,
                                                     type_id timestamp_type_id,
                                                     parquet::Type physical,
                                                     int8_t converted,
                                                     int32_t length)
{
  int32_t type_width = (physical == parquet::FIXED_LEN_BYTE_ARRAY) ? length : 0;
  int32_t clock_rate = 0;
  if (column_type_id == type_id::INT8 or column_type_id == type_id::UINT8) {
    type_width = 1;  // I32 -> I8
  } else if (column_type_id == type_id::INT16 or column_type_id == type_id::UINT16) {
    type_width = 2;  // I32 -> I16
  } else if (column_type_id == type_id::INT32) {
    type_width = 4;  // str -> hash32
  } else if (is_timestamp(data_type{column_type_id})) {
    clock_rate = to_clockrate(timestamp_type_id);
  }

  int8_t converted_type = converted;
  if (converted_type == parquet::DECIMAL && column_type_id != type_id::FLOAT64) {
    converted_type = parquet::UNKNOWN;  // Not converting to float64
  }

  return std::make_tuple(type_width, clock_rate, converted_type);
}

}  // namespace

std::string name_from_path(const std::vector<std::string> &path_in_schema)
{
  std::string s = (path_in_schema.size() > 0) ? path_in_schema[0] : "";
  for (size_t i = 1; i < path_in_schema.size(); i++) { s += "." + path_in_schema[i]; }
  return s;
}

/**
 * @brief Class for parsing dataset metadata
 */
struct metadata : public FileMetaData {
  explicit metadata(datasource *source)
  {
    constexpr auto header_len = sizeof(file_header_s);
    constexpr auto ender_len  = sizeof(file_ender_s);

    const auto len           = source->size();
    const auto header_buffer = source->host_read(0, header_len);
    const auto header        = (const file_header_s *)header_buffer->data();
    const auto ender_buffer  = source->host_read(len - ender_len, ender_len);
    const auto ender         = (const file_ender_s *)ender_buffer->data();
    CUDF_EXPECTS(len > header_len + ender_len, "Incorrect data source");
    CUDF_EXPECTS(header->magic == PARQUET_MAGIC && ender->magic == PARQUET_MAGIC,
                 "Corrupted header or footer");
    CUDF_EXPECTS(ender->footer_len != 0 && ender->footer_len <= (len - header_len - ender_len),
                 "Incorrect footer length");

    const auto buffer = source->host_read(len - ender->footer_len - ender_len, ender->footer_len);
    CompactProtocolReader cp(buffer->data(), ender->footer_len);
    CUDF_EXPECTS(cp.read(this), "Cannot parse metadata");
    CUDF_EXPECTS(cp.InitSchema(this), "Cannot initialize schema");
  }
};

class aggregate_metadata {
  std::vector<metadata> const per_file_metadata;
  std::map<std::string, std::string> const agg_keyval_map;
  size_type const num_rows;
  size_type const num_row_groups;
  std::vector<std::string> const column_names;

  /**
   * @brief Create a metadata object from each element in the source vector
   */
  auto metadatas_from_sources(std::vector<std::unique_ptr<datasource>> const &sources)
  {
    std::vector<metadata> metadatas;
    std::transform(
      sources.cbegin(), sources.cend(), std::back_inserter(metadatas), [](auto const &source) {
        return metadata(source.get());
      });
    return metadatas;
  }

  /**
   * @brief Merge the keyvalue maps from each per-file metadata object into a single map.
   */
  auto merge_keyval_metadata()
  {
    std::map<std::string, std::string> merged;
    // merge key/value maps TODO: warn/throw if there are mismatches?
    for (auto const &pfm : per_file_metadata) {
      for (auto const &kv : pfm.key_value_metadata) { merged[kv.key] = kv.value; }
    }
    return merged;
  }

  /**
   * @brief Sums up the number of rows of each source
   */
  size_type calc_num_rows() const
  {
    return std::accumulate(
      per_file_metadata.begin(), per_file_metadata.end(), 0, [](auto &sum, auto &pfm) {
        return sum + pfm.num_rows;
      });
  }

  /**
   * @brief Sums up the number of row groups of each source
   */
  size_type calc_num_row_groups() const
  {
    return std::accumulate(
      per_file_metadata.begin(), per_file_metadata.end(), 0, [](auto &sum, auto &pfm) {
        return sum + pfm.row_groups.size();
      });
  }
  std::vector<std::string> gather_column_names()
  {
    for (auto const &pfm : per_file_metadata) {
      if (pfm.row_groups.size() != 0) {
        std::vector<std::string> column_names;
        for (const auto &chunk : pfm.row_groups[0].columns) {
          column_names.emplace_back(name_from_path(chunk.meta_data.path_in_schema));
        }
        return column_names;
      }
    }
    return {};
  }

 public:
  aggregate_metadata(std::vector<std::unique_ptr<datasource>> const &sources)
    : per_file_metadata(metadatas_from_sources(sources)),
      agg_keyval_map(merge_keyval_metadata()),
      num_rows(calc_num_rows()),
      num_row_groups(calc_num_row_groups()),
      column_names(gather_column_names())
  {
    // Verify that the input files have matching numbers of columns
    size_type num_cols = -1;
    for (auto const &pfm : per_file_metadata) {
      if (pfm.row_groups.size() != 0) {
        if (num_cols == -1)
          num_cols = pfm.row_groups[0].columns.size();
        else
          CUDF_EXPECTS(num_cols == static_cast<size_type>(pfm.row_groups[0].columns.size()),
                       "All sources must have the same number of columns");
      }
    }
    // Verify that the input files have matching schemas
    for (auto const &pfm : per_file_metadata) {
      CUDF_EXPECTS(per_file_metadata[0].schema == pfm.schema,
                   "All sources must have the same schemas");
    }
  }

  auto const &get_row_group(size_type idx, size_type src_idx) const
  {
    CUDF_EXPECTS(src_idx >= 0 && src_idx < static_cast<size_type>(per_file_metadata.size()),
                 "invalid source index");
    return per_file_metadata[src_idx].row_groups[idx];
  }

  auto get_num_rows() const { return num_rows; }

  auto get_num_row_groups() const { return num_row_groups; }

  auto const &get_schema(int idx) const { return per_file_metadata[0].schema[idx]; }

  auto const &get_key_value_metadata() const { return agg_keyval_map; }

  /**
   * @brief Extracts the pandas "index_columns" section
   *
   * PANDAS adds its own metadata to the key_value section when writing out the
   * dataframe to a file to aid in exact reconstruction. The JSON-formatted
   * metadata contains the index column(s) and PANDA-specific datatypes.
   *
   * @return comma-separated index column names in quotes
   */
  std::string get_pandas_index() const
  {
    auto it = agg_keyval_map.find("pandas");
    if (it != agg_keyval_map.end()) {
      // Captures a list of quoted strings found inside square brackets after `"index_columns":`
      // Inside quotes supports newlines, brackets, escaped quotes, etc.
      // One-liner regex:
      // "index_columns"\s*:\s*\[\s*((?:"(?:|(?:.*?(?![^\\]")).?)[^\\]?",?\s*)*)\]
      // Documented below.
      std::regex index_columns_expr{
        R"("index_columns"\s*:\s*\[\s*)"  // match preamble, opening square bracket, whitespace
        R"(()"                            // Open first capturing group
        R"((?:")"                         // Open non-capturing group match opening quote
        R"((?:|(?:.*?(?![^\\]")).?))"     // match empty string or anything between quotes
        R"([^\\]?")"                      // Match closing non-escaped quote
        R"(,?\s*)"                        // Match optional comma and whitespace
        R"()*)"                           // Close non-capturing group and repeat 0 or more times
        R"())"                            // Close first capturing group
        R"(\])"                           // Match closing square brackets
      };
      std::smatch sm;
      if (std::regex_search(it->second, sm, index_columns_expr)) { return std::move(sm[1].str()); }
    }
    return "";
  }

  /**
   * @brief Extracts the column name(s) used for the row indexes in a dataframe
   *
   * @param names List of column names to load, where index column name(s) will be added
   */
  void add_pandas_index_names(std::vector<std::string> &names) const
  {
    auto str = get_pandas_index();
    if (str.length() != 0) {
      std::regex index_name_expr{R"(\"((?:\\.|[^\"])*)\")"};
      std::smatch sm;
      while (std::regex_search(str, sm, index_name_expr)) {
        if (sm.size() == 2) {  // 2 = whole match, first item
          if (std::find(names.begin(), names.end(), sm[1].str()) == names.end()) {
            std::regex esc_quote{R"(\\")"};
            names.emplace_back(std::move(std::regex_replace(sm[1].str(), esc_quote, R"(")")));
          }
        }
        str = sm.suffix();
      }
    }
  }

  struct row_group_info {
    size_type const index;
    size_t const start_row;  // TODO source index
    size_type const source_index;
    row_group_info(size_type index, size_t start_row, size_type source_index)
      : index(index), start_row(start_row), source_index(source_index)
    {
    }
  };

  /**
   * @brief Filters and reduces down to a selection of row groups
   *
   * @param row_groups Lists of row group to reads, one per source
   * @param row_start Starting row of the selection
   * @param row_count Total number of rows selected
   *
   * @return List of row group indexes and its starting row
   */
  auto select_row_groups(std::vector<std::vector<size_type>> const &row_groups,
                         size_type &row_start,
                         size_type &row_count) const
  {
    if (!row_groups.empty()) {
      std::vector<row_group_info> selection;
      CUDF_EXPECTS(row_groups.size() == per_file_metadata.size(),
                   "Must specify row groups for each source");

      row_count = 0;
      for (size_t src_idx = 0; src_idx < row_groups.size(); ++src_idx) {
        for (auto const &rowgroup_idx : row_groups[src_idx]) {
          CUDF_EXPECTS(
            rowgroup_idx >= 0 &&
              rowgroup_idx < static_cast<size_type>(per_file_metadata[src_idx].row_groups.size()),
            "Invalid rowgroup index");
          selection.emplace_back(rowgroup_idx, row_count, src_idx);
          row_count += get_row_group(rowgroup_idx, src_idx).num_rows;
        }
      }
      return selection;
    }

    row_start = std::max(row_start, 0);
    if (row_count < 0) {
      row_count = static_cast<size_type>(
        std::min<int64_t>(get_num_rows(), std::numeric_limits<size_type>::max()));
    }
    CUDF_EXPECTS(row_count >= 0, "Invalid row count");
    CUDF_EXPECTS(row_start <= get_num_rows(), "Invalid row start");

    std::vector<row_group_info> selection;
    size_type count = 0;
    for (size_t src_idx = 0; src_idx < per_file_metadata.size(); ++src_idx) {
      for (size_t rg_idx = 0; rg_idx < per_file_metadata[src_idx].row_groups.size(); ++rg_idx) {
        auto const chunk_start_row = count;
        count += get_row_group(rg_idx, src_idx).num_rows;
        if (count > row_start || count == 0) {
          selection.emplace_back(rg_idx, chunk_start_row, src_idx);
        }
        if (count >= row_start + row_count) { break; }
      }
    }
    return selection;
  }

  /**
   * @brief Filters and reduces down to a selection of columns
   *
   * @param use_names List of column names to select
   * @param include_index Whether to always include the PANDAS index column(s)
   *
   * @return List of column names
   */
  auto select_columns(std::vector<std::string> use_names, bool include_index) const
  {
    std::vector<std::pair<int, std::string>> selection;
    if (use_names.empty()) {
      // No columns specified; include all in the dataset
      for (const auto &name : column_names) { selection.emplace_back(selection.size(), name); }
    } else {
      // Load subset of columns; include PANDAS index unless excluded
      if (include_index) { add_pandas_index_names(use_names); }
      for (const auto &use_name : use_names) {
        for (size_t i = 0; i < column_names.size(); ++i) {
          if (column_names[i] == use_name) {
            selection.emplace_back(i, column_names[i]);
            break;
          }
        }
      }
    }

    return selection;
  }
};

void reader::impl::read_column_chunks(
  std::vector<rmm::device_buffer> &page_data,
  hostdevice_vector<gpu::ColumnChunkDesc> &chunks,  // TODO const?
  size_t begin_chunk,
  size_t end_chunk,
  const std::vector<size_t> &column_chunk_offsets,
  std::vector<size_type> const &chunk_source_map,
  hipStream_t stream)
{
  // Transfer chunk data, coalescing adjacent chunks
  for (size_t chunk = begin_chunk; chunk < end_chunk;) {
    const size_t io_offset   = column_chunk_offsets[chunk];
    size_t io_size           = chunks[chunk].compressed_size;
    size_t next_chunk        = chunk + 1;
    const bool is_compressed = (chunks[chunk].codec != parquet::Compression::UNCOMPRESSED);
    while (next_chunk < end_chunk) {
      const size_t next_offset = column_chunk_offsets[next_chunk];
      const bool is_next_compressed =
        (chunks[next_chunk].codec != parquet::Compression::UNCOMPRESSED);
      if (next_offset != io_offset + io_size || is_next_compressed != is_compressed) {
        // Can't merge if not contiguous or mixing compressed and uncompressed
        // Not coalescing uncompressed with compressed chunks is so that compressed buffers can be
        // freed earlier (immediately after decompression stage) to limit peak memory requirements
        break;
      }
      io_size += chunks[next_chunk].compressed_size;
      next_chunk++;
    }
    if (io_size != 0) {
      auto buffer         = _sources[chunk_source_map[chunk]]->host_read(io_offset, io_size);
      page_data[chunk]    = rmm::device_buffer(buffer->data(), buffer->size(), stream);
      uint8_t *d_compdata = reinterpret_cast<uint8_t *>(page_data[chunk].data());
      do {
        chunks[chunk].compressed_data = d_compdata;
        d_compdata += chunks[chunk].compressed_size;
      } while (++chunk != next_chunk);
    } else {
      chunk = next_chunk;
    }
  }
}

size_t reader::impl::count_page_headers(hostdevice_vector<gpu::ColumnChunkDesc> &chunks,
                                        hipStream_t stream)
{
  size_t total_pages = 0;

  CUDA_TRY(hipMemcpyAsync(
    chunks.device_ptr(), chunks.host_ptr(), chunks.memory_size(), hipMemcpyHostToDevice, stream));
  CUDA_TRY(gpu::DecodePageHeaders(chunks.device_ptr(), chunks.size(), stream));
  CUDA_TRY(hipMemcpyAsync(
    chunks.host_ptr(), chunks.device_ptr(), chunks.memory_size(), hipMemcpyDeviceToHost, stream));
  CUDA_TRY(hipStreamSynchronize(stream));

  for (size_t c = 0; c < chunks.size(); c++) {
    total_pages += chunks[c].num_data_pages + chunks[c].num_dict_pages;
  }

  return total_pages;
}

void reader::impl::decode_page_headers(hostdevice_vector<gpu::ColumnChunkDesc> &chunks,
                                       hostdevice_vector<gpu::PageInfo> &pages,
                                       hipStream_t stream)
{
  for (size_t c = 0, page_count = 0; c < chunks.size(); c++) {
    chunks[c].max_num_pages = chunks[c].num_data_pages + chunks[c].num_dict_pages;
    chunks[c].page_info     = pages.device_ptr(page_count);
    page_count += chunks[c].max_num_pages;
  }

  CUDA_TRY(hipMemcpyAsync(
    chunks.device_ptr(), chunks.host_ptr(), chunks.memory_size(), hipMemcpyHostToDevice, stream));
  CUDA_TRY(gpu::DecodePageHeaders(chunks.device_ptr(), chunks.size(), stream));
  CUDA_TRY(hipMemcpyAsync(
    pages.host_ptr(), pages.device_ptr(), pages.memory_size(), hipMemcpyDeviceToHost, stream));
  CUDA_TRY(hipStreamSynchronize(stream));
}

rmm::device_buffer reader::impl::decompress_page_data(
  hostdevice_vector<gpu::ColumnChunkDesc> &chunks,
  hostdevice_vector<gpu::PageInfo> &pages,
  hipStream_t stream)
{
  auto for_each_codec_page = [&](parquet::Compression codec, const std::function<void(size_t)> &f) {
    for (size_t c = 0, page_count = 0; c < chunks.size(); c++) {
      const auto page_stride = chunks[c].max_num_pages;
      if (chunks[c].codec == codec) {
        for (int k = 0; k < page_stride; k++) { f(page_count + k); }
      }
      page_count += page_stride;
    }
  };

  // Brotli scratch memory for decompressing
  rmm::device_vector<uint8_t> debrotli_scratch;

  // Count the exact number of compressed pages
  size_t num_comp_pages    = 0;
  size_t total_decomp_size = 0;
  std::array<std::pair<parquet::Compression, size_t>, 3> codecs{std::make_pair(parquet::GZIP, 0),
                                                                std::make_pair(parquet::SNAPPY, 0),
                                                                std::make_pair(parquet::BROTLI, 0)};

  for (auto &codec : codecs) {
    for_each_codec_page(codec.first, [&](size_t page) {
      total_decomp_size += pages[page].uncompressed_page_size;
      codec.second++;
      num_comp_pages++;
    });
    if (codec.first == parquet::BROTLI && codec.second > 0) {
      debrotli_scratch.resize(get_gpu_debrotli_scratch_size(codec.second));
    }
  }

  // Dispatch batches of pages to decompress for each codec
  rmm::device_buffer decomp_pages(total_decomp_size, stream);
  hostdevice_vector<gpu_inflate_input_s> inflate_in(0, num_comp_pages, stream);
  hostdevice_vector<gpu_inflate_status_s> inflate_out(0, num_comp_pages, stream);

  size_t decomp_offset = 0;
  int32_t argc         = 0;
  for (const auto &codec : codecs) {
    if (codec.second > 0) {
      int32_t start_pos = argc;

      for_each_codec_page(codec.first, [&](size_t page) {
        auto dst_base              = static_cast<uint8_t *>(decomp_pages.data());
        inflate_in[argc].srcDevice = pages[page].page_data;
        inflate_in[argc].srcSize   = pages[page].compressed_page_size;
        inflate_in[argc].dstDevice = dst_base + decomp_offset;
        inflate_in[argc].dstSize   = pages[page].uncompressed_page_size;

        inflate_out[argc].bytes_written = 0;
        inflate_out[argc].status        = static_cast<uint32_t>(-1000);
        inflate_out[argc].reserved      = 0;

        pages[page].page_data = (uint8_t *)inflate_in[argc].dstDevice;
        decomp_offset += inflate_in[argc].dstSize;
        argc++;
      });

      CUDA_TRY(hipMemcpyAsync(inflate_in.device_ptr(start_pos),
                               inflate_in.host_ptr(start_pos),
                               sizeof(decltype(inflate_in)::value_type) * (argc - start_pos),
                               hipMemcpyHostToDevice,
                               stream));
      CUDA_TRY(hipMemcpyAsync(inflate_out.device_ptr(start_pos),
                               inflate_out.host_ptr(start_pos),
                               sizeof(decltype(inflate_out)::value_type) * (argc - start_pos),
                               hipMemcpyHostToDevice,
                               stream));
      switch (codec.first) {
        case parquet::GZIP:
          CUDA_TRY(gpuinflate(inflate_in.device_ptr(start_pos),
                              inflate_out.device_ptr(start_pos),
                              argc - start_pos,
                              1,
                              stream))
          break;
        case parquet::SNAPPY:
          CUDA_TRY(gpu_unsnap(inflate_in.device_ptr(start_pos),
                              inflate_out.device_ptr(start_pos),
                              argc - start_pos,
                              stream));
          break;
        case parquet::BROTLI:
          CUDA_TRY(gpu_debrotli(inflate_in.device_ptr(start_pos),
                                inflate_out.device_ptr(start_pos),
                                debrotli_scratch.data().get(),
                                debrotli_scratch.size(),
                                argc - start_pos,
                                stream));
          break;
        default: CUDF_EXPECTS(false, "Unexpected decompression dispatch"); break;
      }
      CUDA_TRY(hipMemcpyAsync(inflate_out.host_ptr(start_pos),
                               inflate_out.device_ptr(start_pos),
                               sizeof(decltype(inflate_out)::value_type) * (argc - start_pos),
                               hipMemcpyDeviceToHost,
                               stream));
    }
  }
  CUDA_TRY(hipStreamSynchronize(stream));

  // Update the page information in device memory with the updated value of
  // page_data; it now points to the uncompressed data buffer
  CUDA_TRY(hipMemcpyAsync(
    pages.device_ptr(), pages.host_ptr(), pages.memory_size(), hipMemcpyHostToDevice, stream));

  return decomp_pages;
}

void reader::impl::decode_page_data(hostdevice_vector<gpu::ColumnChunkDesc> &chunks,
                                    hostdevice_vector<gpu::PageInfo> &pages,
                                    size_t min_row,
                                    size_t total_rows,
                                    const std::vector<int> &chunk_col_map,
                                    std::vector<column_buffer> &out_buffers,
                                    hipStream_t stream)
{
  auto is_dict_chunk = [](const gpu::ColumnChunkDesc &chunk) {
    return (chunk.data_type & 0x7) == BYTE_ARRAY && chunk.num_dict_pages > 0;
  };

  // Count the number of string dictionary entries
  // NOTE: Assumes first page in the chunk is always the dictionary page
  size_t total_str_dict_indexes = 0;
  for (size_t c = 0, page_count = 0; c < chunks.size(); c++) {
    if (is_dict_chunk(chunks[c])) { total_str_dict_indexes += pages[page_count].num_values; }
    page_count += chunks[c].max_num_pages;
  }

  // Build index for string dictionaries since they can't be indexed
  // directly due to variable-sized elements
  rmm::device_vector<gpu::nvstrdesc_s> str_dict_index;
  if (total_str_dict_indexes > 0) { str_dict_index.resize(total_str_dict_indexes); }

  // Update chunks with pointers to column data
  for (size_t c = 0, page_count = 0, str_ofs = 0; c < chunks.size(); c++) {
    if (is_dict_chunk(chunks[c])) {
      chunks[c].str_dict_index = str_dict_index.data().get() + str_ofs;
      str_ofs += pages[page_count].num_values;
    }
    chunks[c].column_data_base = out_buffers[chunk_col_map[c]].data();
    chunks[c].valid_map_base   = out_buffers[chunk_col_map[c]].null_mask();
    page_count += chunks[c].max_num_pages;
  }

  CUDA_TRY(hipMemcpyAsync(
    chunks.device_ptr(), chunks.host_ptr(), chunks.memory_size(), hipMemcpyHostToDevice, stream));
  if (total_str_dict_indexes > 0) {
    CUDA_TRY(gpu::BuildStringDictionaryIndex(chunks.device_ptr(), chunks.size(), stream));
  }
  CUDA_TRY(gpu::DecodePageData(pages.device_ptr(),
                               pages.size(),
                               chunks.device_ptr(),
                               chunks.size(),
                               total_rows,
                               min_row,
                               stream));
  CUDA_TRY(hipMemcpyAsync(
    pages.host_ptr(), pages.device_ptr(), pages.memory_size(), hipMemcpyDeviceToHost, stream));
  CUDA_TRY(hipStreamSynchronize(stream));

  for (size_t i = 0; i < pages.size(); i++) {
    if (pages[i].num_rows > 0) {
      const size_t c = pages[i].chunk_idx;
      if (c < chunks.size()) {
        out_buffers[chunk_col_map[c]].null_count() += pages[i].num_rows - pages[i].valid_count;
      }
    }
  }
}

reader::impl::impl(std::vector<std::unique_ptr<datasource>> &&sources,
                   reader_options const &options,
                   rmm::mr::device_memory_resource *mr)
  : _sources(std::move(sources)), _mr(mr)
{
  // Open and parse the source dataset metadata
  _metadata = std::make_unique<aggregate_metadata>(_sources);

  // Select only columns required by the options
  _selected_columns = _metadata->select_columns(options.columns, options.use_pandas_metadata);

  // Override output timestamp resolution if requested
  if (options.timestamp_type.id() != type_id::EMPTY) { _timestamp_type = options.timestamp_type; }

  // Strings may be returned as either string or categorical columns
  _strings_to_categorical = options.strings_to_categorical;
}

table_with_metadata reader::impl::read(size_type skip_rows,
                                       size_type num_rows,
                                       std::vector<std::vector<size_type>> const &row_group_list,
                                       hipStream_t stream)
{
  // Select only row groups required
  const auto selected_row_groups =
    _metadata->select_row_groups(row_group_list, skip_rows, num_rows);

  // Get a list of column data types
  std::vector<data_type> column_types;
  if (_metadata->get_num_row_groups() != 0) {
    for (const auto &col : _selected_columns) {
      auto const &col_schema =
        _metadata->get_schema(_metadata->get_row_group(0, 0).columns[col.first].schema_idx);
      auto const col_type = to_type_id(col_schema.type,
                                       col_schema.converted_type,
                                       _strings_to_categorical,
                                       _timestamp_type.id(),
                                       col_schema.decimal_scale);
      CUDF_EXPECTS(col_type != type_id::EMPTY, "Unknown type");
      column_types.emplace_back(col_type);
    }
  }

  std::vector<std::unique_ptr<column>> out_columns;
  out_columns.reserve(column_types.size());

  if (selected_row_groups.size() != 0 && column_types.size() != 0) {
    // Descriptors for all the chunks that make up the selected columns
    const auto num_columns = _selected_columns.size();
    const auto num_chunks  = selected_row_groups.size() * num_columns;
    hostdevice_vector<gpu::ColumnChunkDesc> chunks(0, num_chunks, stream);

    // Association between each column chunk and its column
    std::vector<int> chunk_col_map(num_chunks);
    // Association between each column chunk and its source
    std::vector<size_type> chunk_source_map(num_chunks);

    // Tracker for eventually deallocating compressed and uncompressed data
    std::vector<rmm::device_buffer> page_data(num_chunks);

    // Keep track of column chunk file offsets
    std::vector<size_t> column_chunk_offsets(num_chunks);

    // Initialize column chunk information
    size_t total_decompressed_size = 0;
    auto remaining_rows            = num_rows;
    for (const auto &rg : selected_row_groups) {
      const auto &row_group       = _metadata->get_row_group(rg.index, rg.source_index);
      auto const row_group_start  = rg.start_row;
      auto const row_group_source = rg.source_index;
      auto const row_group_rows   = std::min<int>(remaining_rows, row_group.num_rows);
      auto const io_chunk_idx     = chunks.size();

      for (size_t i = 0; i < num_columns; ++i) {
        auto const col         = _selected_columns[i];
        auto const &col_meta   = row_group.columns[col.first].meta_data;
        auto const &col_schema = _metadata->get_schema(row_group.columns[col.first].schema_idx);

        // Spec requires each row group to contain exactly one chunk for every
        // column. If there are too many or too few, continue with best effort
        if (col.second != name_from_path(col_meta.path_in_schema)) {
          std::cerr << "Detected mismatched column chunk" << std::endl;
          continue;
        }
        if (chunks.size() >= chunks.max_size()) {
          std::cerr << "Detected too many column chunks" << std::endl;
          continue;
        }

        int32_t type_width;
        int32_t clock_rate;
        int8_t converted_type;
        std::tie(type_width, clock_rate, converted_type) =
          conversion_info(column_types[i].id(),
                          _timestamp_type.id(),
                          col_schema.type,
                          col_schema.converted_type,
                          col_schema.type_length);

        column_chunk_offsets[chunks.size()] =
          (col_meta.dictionary_page_offset != 0)
            ? std::min(col_meta.data_page_offset, col_meta.dictionary_page_offset)
            : col_meta.data_page_offset;

        chunks.insert(gpu::ColumnChunkDesc(col_meta.total_compressed_size,
                                           nullptr,
                                           col_meta.num_values,
                                           col_schema.type,
                                           type_width,
                                           row_group_start,
                                           row_group_rows,
                                           col_schema.max_definition_level,
                                           col_schema.max_repetition_level,
                                           required_bits(col_schema.max_definition_level),
                                           required_bits(col_schema.max_repetition_level),
                                           col_meta.codec,
                                           converted_type,
                                           col_schema.decimal_scale,
                                           clock_rate));

        // Map each column chunk to its column index and its source index
        chunk_col_map[chunks.size() - 1]    = i;
        chunk_source_map[chunks.size() - 1] = row_group_source;

        if (col_meta.codec != Compression::UNCOMPRESSED) {
          total_decompressed_size += col_meta.total_uncompressed_size;
        }
      }
      // Read compressed chunk data to device memory
      read_column_chunks(page_data,
                         chunks,
                         io_chunk_idx,
                         chunks.size(),
                         column_chunk_offsets,
                         chunk_source_map,
                         stream);

      remaining_rows -= row_group.num_rows;
    }
    assert(remaining_rows <= 0);

    // Process dataset chunk pages into output columns
    const auto total_pages = count_page_headers(chunks, stream);
    if (total_pages > 0) {
      hostdevice_vector<gpu::PageInfo> pages(total_pages, total_pages, stream);
      rmm::device_buffer decomp_page_data;

      decode_page_headers(chunks, pages, stream);
      if (total_decompressed_size > 0) {
        decomp_page_data = decompress_page_data(chunks, pages, stream);
        // Free compressed data
        for (size_t c = 0; c < chunks.size(); c++) {
          if (chunks[c].codec != parquet::Compression::UNCOMPRESSED && page_data[c].size() != 0) {
            page_data[c].resize(0);
            page_data[c].shrink_to_fit();
          }
        }
      }

      std::vector<column_buffer> out_buffers;
      out_buffers.reserve(column_types.size());
      for (size_t i = 0; i < column_types.size(); ++i) {
        auto col                    = _selected_columns[i];
        auto const &first_row_group = _metadata->get_row_group(selected_row_groups[0].index,
                                                               selected_row_groups[0].source_index);
        auto &col_schema = _metadata->get_schema(first_row_group.columns[col.first].schema_idx);
        bool is_nullable = (col_schema.max_definition_level != 0);
        out_buffers.emplace_back(column_types[i], num_rows, is_nullable, stream, _mr);
      }

      decode_page_data(chunks, pages, skip_rows, num_rows, chunk_col_map, out_buffers, stream);

      for (size_t i = 0; i < column_types.size(); ++i) {
        out_columns.emplace_back(
          make_column(column_types[i], num_rows, out_buffers[i], stream, _mr));
      }
    }
  }

  // Create empty columns as needed
  for (size_t i = out_columns.size(); i < column_types.size(); ++i) {
    out_columns.emplace_back(make_empty_column(column_types[i]));
  }

  table_metadata out_metadata;
  // Return column names (must match order of returned columns)
  out_metadata.column_names.resize(_selected_columns.size());
  for (size_t i = 0; i < _selected_columns.size(); i++) {
    out_metadata.column_names[i] = _selected_columns[i].second;
  }
  // Return user metadata
  out_metadata.user_data = _metadata->get_key_value_metadata();

  return {std::make_unique<table>(std::move(out_columns)), std::move(out_metadata)};
}

// Forward to implementation
reader::reader(std::vector<std::string> const &filepaths,
               reader_options const &options,
               rmm::mr::device_memory_resource *mr)
  : _impl(std::make_unique<impl>(datasource::create(filepaths), options, mr))
{
}

// Forward to implementation
reader::reader(std::vector<std::unique_ptr<cudf::io::datasource>> &&sources,
               reader_options const &options,
               rmm::mr::device_memory_resource *mr)
  : _impl(std::make_unique<impl>(std::move(sources), options, mr))
{
}

// Destructor within this translation unit
reader::~reader() = default;

// Forward to implementation
table_with_metadata reader::read_all(hipStream_t stream) { return _impl->read(0, -1, {}, stream); }

// Forward to implementation
table_with_metadata reader::read_row_groups(std::vector<std::vector<size_type>> const &row_groups,
                                            hipStream_t stream)
{
  return _impl->read(0, -1, row_groups, stream);
}

// Forward to implementation
table_with_metadata reader::read_rows(size_type skip_rows, size_type num_rows, hipStream_t stream)
{
  return _impl->read(skip_rows, (num_rows != 0) ? num_rows : -1, {}, stream);
}

}  // namespace parquet
}  // namespace detail
}  // namespace io
}  // namespace cudf
