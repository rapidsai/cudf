#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

/**
 * @file reader_impl.cu
 * @brief cuDF-IO Parquet reader class implementation
 */

#include "reader_impl.hpp"

#include <io/comp/gpuinflate.h>

#include <cudf/table/table.hpp>
#include <cudf/utilities/error.hpp>
#include <cudf/utilities/traits.hpp>

#include <rmm/thrust_rmm_allocator.h>
#include <rmm/device_buffer.hpp>

#include <algorithm>
#include <array>

namespace cudf {
namespace experimental {
namespace io {
namespace detail {
namespace parquet {

// Import functionality that's independent of legacy code
using namespace cudf::io::parquet;
using namespace cudf::io;

namespace {

/**
 * @brief Function that translates Parquet datatype to cuDF type enum
 */
constexpr type_id to_type_id(parquet::Type physical,
                             parquet::ConvertedType logical,
                             bool strings_to_categorical,
                             type_id timestamp_type_id, int32_t decimal_scale) {
  // Logical type used for actual data interpretation; the legacy converted type
  // is superceded by 'logical' type whenever available.
  switch (logical) {
    case parquet::UINT_8:
    case parquet::INT_8:
      return type_id::INT8;
    case parquet::UINT_16:
    case parquet::INT_16:
      return type_id::INT16;
    case parquet::DATE:
      return type_id::TIMESTAMP_DAYS;
    case parquet::TIMESTAMP_MICROS:
      return (timestamp_type_id != type_id::EMPTY)
                 ? timestamp_type_id
                 : type_id::TIMESTAMP_MICROSECONDS;
    case parquet::TIMESTAMP_MILLIS:
      return (timestamp_type_id != type_id::EMPTY)
                 ? timestamp_type_id
                 : type_id::TIMESTAMP_MILLISECONDS;
    case parquet::DECIMAL:
      if (decimal_scale != 0 ||
          (physical != parquet::INT32 && physical != parquet::INT64)) {
        return type_id::FLOAT64;
      }
      break;
    default:
      break;
  }

  // Physical storage type supported by Parquet; controls the on-disk storage
  // format in combination with the encoding type.
  switch (physical) {
    case parquet::BOOLEAN:
      return type_id::BOOL8;
    case parquet::INT32:
      return type_id::INT32;
    case parquet::INT64:
      return type_id::INT64;
    case parquet::FLOAT:
      return type_id::FLOAT32;
    case parquet::DOUBLE:
      return type_id::FLOAT64;
    case parquet::BYTE_ARRAY:
    case parquet::FIXED_LEN_BYTE_ARRAY:
      // Can be mapped to GDF_CATEGORY (32-bit hash) or GDF_STRING (nvstring)
      return strings_to_categorical ? type_id::INT32 : type_id::STRING;
    case parquet::INT96:
      return (timestamp_type_id != type_id::EMPTY)
                 ? timestamp_type_id
                 : type_id::TIMESTAMP_NANOSECONDS;
    default:
      break;
  }

  return type_id::EMPTY;
}

/**
 * @brief Function that translates cuDF time unit to Parquet clock frequency
 */
constexpr int32_t to_clockrate(type_id timestamp_type_id) {
  switch (timestamp_type_id) {
    case type_id::TIMESTAMP_SECONDS:
      return 1;
    case type_id::TIMESTAMP_MILLISECONDS:
      return 1000;
    case type_id::TIMESTAMP_MICROSECONDS:
      return 1000000;
    case type_id::TIMESTAMP_NANOSECONDS:
      return 1000000000;
    default:
      return 0;
  }
}

/**
 * @brief Function that returns the required the number of bits to store a value
 */
template <typename T = uint8_t>
T required_bits(uint32_t max_level) {
  return static_cast<T>(CompactProtocolReader::NumRequiredBits(max_level));
}

std::tuple<int32_t, int32_t, int8_t> conversion_info(type_id column_type_id,
                                                     type_id timestamp_type_id,
                                                     parquet::Type physical,
                                                     int8_t converted,
                                                     int32_t length) {
  int32_t type_width = (physical == parquet::FIXED_LEN_BYTE_ARRAY) ? length : 0;
  int32_t clock_rate = 0;
  if (column_type_id == type_id::INT8) {
    type_width = 1;  // I32 -> I8
  } else if (column_type_id == type_id::INT16) {
    type_width = 2;  // I32 -> I16
  } else if (column_type_id == type_id::INT32) {
    type_width = 4;  // str -> hash32
  } else if (is_timestamp(data_type{column_type_id})) {
    clock_rate = to_clockrate(timestamp_type_id);
  }

  int8_t converted_type = converted;
  if (converted_type == parquet::DECIMAL &&
      column_type_id != type_id::FLOAT64) {
    converted_type = parquet::UNKNOWN;  // Not converting to float64
  }

  return std::make_tuple(type_width, clock_rate, converted_type);
}

}  // namespace

/**
 * @brief Class for parsing dataset metadata
 */
struct metadata : public FileMetaData {
  explicit metadata(datasource *source) {
    constexpr auto header_len = sizeof(file_header_s);
    constexpr auto ender_len = sizeof(file_ender_s);

    const auto len = source->size();
    const auto header_buffer = source->get_buffer(0, header_len);
    const auto header = (const file_header_s *)header_buffer->data();
    const auto ender_buffer = source->get_buffer(len - ender_len, ender_len);
    const auto ender = (const file_ender_s *)ender_buffer->data();
    CUDF_EXPECTS(len > header_len + ender_len, "Incorrect data source");
    CUDF_EXPECTS(
        header->magic == PARQUET_MAGIC && ender->magic == PARQUET_MAGIC,
        "Corrupted header or footer");
    CUDF_EXPECTS(ender->footer_len != 0 &&
                     ender->footer_len <= (len - header_len - ender_len),
                 "Incorrect footer length");

    const auto buffer = source->get_buffer(len - ender->footer_len - ender_len,
                                           ender->footer_len);
    CompactProtocolReader cp(buffer->data(), ender->footer_len);
    CUDF_EXPECTS(cp.read(this), "Cannot parse metadata");
    CUDF_EXPECTS(cp.InitSchema(this), "Cannot initialize schema");
  }

  inline int get_total_rows() const { return num_rows; }
  inline int get_num_row_groups() const { return row_groups.size(); }
  inline int get_num_columns() const { return row_groups[0].columns.size(); }

  std::string get_column_name(const std::vector<std::string> &path_in_schema) {
    std::string s = (path_in_schema.size() > 0) ? path_in_schema[0] : "";
    for (size_t i = 1; i < path_in_schema.size(); i++) {
      s += "." + path_in_schema[i];
    }
    return s;
  }

  std::vector<std::string> get_column_names() {
    std::vector<std::string> all_names;
    for (const auto &chunk : row_groups[0].columns) {
      all_names.emplace_back(get_column_name(chunk.meta_data.path_in_schema));
    }
    return all_names;
  }

  /**
   * @brief Extracts the column name used for the row indexes in a dataframe
   *
   * PANDAS adds its own metadata to the key_value section when writing out the
   * dataframe to a file to aid in exact reconstruction. The JSON-formatted
   * metadata contains the index column(s) and PANDA-specific datatypes.
   *
   * @return std::string Name of the index column
   */
  std::string get_pandas_index_name() {
    auto it =
        std::find_if(key_value_metadata.begin(), key_value_metadata.end(),
                     [](const auto &item) { return item.key == "pandas"; });

    if (it != key_value_metadata.end()) {
      const auto pos = it->value.find("index_columns");
      if (pos != std::string::npos) {
        const auto begin = it->value.find('[', pos);
        const auto end = it->value.find(']', begin);
        if ((end - begin) > 1) {
          return it->value.substr(begin + 2, end - begin - 3);
        }
      }
    }
    return "";
  }

  /**
   * @brief Filters and reduces down to a selection of row groups
   *
   * @param row_group Index of the row group to select
   * @param row_start Starting row of the selection
   * @param row_count Total number of rows selected
   *
   * @return List of row group indexes and its starting row
   */
  auto select_row_groups(int row_group, int &row_start, int &row_count) {
    std::vector<std::pair<int, int>> selection;

    if (row_group != -1) {
      CUDF_EXPECTS(row_group < get_num_row_groups(), "Non-existent row group");
      for (int i = 0; i < row_group; ++i) {
        row_start += row_groups[i].num_rows;
      }
      selection.emplace_back(row_group, row_start);
      row_count = row_groups[row_group].num_rows;
    } else {
      row_start = std::max(row_start, 0);
      if (row_count == -1) {
        row_count = get_total_rows();
      }
      CUDF_EXPECTS(row_count >= 0, "Invalid row count");
      CUDF_EXPECTS(row_start <= get_total_rows(), "Invalid row start");

      for (int i = 0, count = 0; i < (int)row_groups.size(); ++i) {
        count += row_groups[i].num_rows;
        if (count > row_start || count == 0) {
          selection.emplace_back(i, count - row_groups[i].num_rows);
        }
        if (count >= (row_start + row_count)) {
          break;
        }
      }
    }

    return selection;
  }

  /**
   * @brief Filters and reduces down to a selection of columns
   *
   * @param use_names List of column names to select
   * @param include_index Whether to always include the PANDAS index column
   * @param pandas_index Name of the PANDAS index column
   *
   * @return List of column names
   */
  auto select_columns(std::vector<std::string> use_names, bool include_index,
                      const std::string &pandas_index) {
    std::vector<std::pair<int, std::string>> selection;

    const auto names = get_column_names();
    if (use_names.empty()) {
      // No columns specified; include all in the dataset
      for (const auto &name : names) {
        selection.emplace_back(selection.size(), name);
      }
    } else {
      // Load subset of columns; include PANDAS index unless excluded
      if (include_index) {
        if (std::find(use_names.begin(), use_names.end(), pandas_index) ==
            use_names.end()) {
          use_names.push_back(pandas_index);
        }
      }
      for (const auto &use_name : use_names) {
        for (size_t i = 0; i < names.size(); ++i) {
          if (names[i] == use_name) {
            selection.emplace_back(i, names[i]);
            break;
          }
        }
      }
    }

    return selection;
  }
};

size_t reader::impl::count_page_headers(
    const hostdevice_vector<gpu::ColumnChunkDesc> &chunks,
    hipStream_t stream) {
  size_t total_pages = 0;

  CUDA_TRY(hipMemcpyAsync(chunks.device_ptr(), chunks.host_ptr(),
                           chunks.memory_size(), hipMemcpyHostToDevice,
                           stream));
  CUDA_TRY(gpu::DecodePageHeaders(chunks.device_ptr(), chunks.size(), stream));
  CUDA_TRY(hipMemcpyAsync(chunks.host_ptr(), chunks.device_ptr(),
                           chunks.memory_size(), hipMemcpyDeviceToHost,
                           stream));
  CUDA_TRY(hipStreamSynchronize(stream));

  for (size_t c = 0; c < chunks.size(); c++) {
    total_pages += chunks[c].num_data_pages + chunks[c].num_dict_pages;
  }

  return total_pages;
}

void reader::impl::decode_page_headers(
    const hostdevice_vector<gpu::ColumnChunkDesc> &chunks,
    const hostdevice_vector<gpu::PageInfo> &pages, hipStream_t stream) {
  for (size_t c = 0, page_count = 0; c < chunks.size(); c++) {
    chunks[c].max_num_pages =
        chunks[c].num_data_pages + chunks[c].num_dict_pages;
    chunks[c].page_info = pages.device_ptr(page_count);
    page_count += chunks[c].max_num_pages;
  }

  CUDA_TRY(hipMemcpyAsync(chunks.device_ptr(), chunks.host_ptr(),
                           chunks.memory_size(), hipMemcpyHostToDevice,
                           stream));
  CUDA_TRY(gpu::DecodePageHeaders(chunks.device_ptr(), chunks.size(), stream));
  CUDA_TRY(hipMemcpyAsync(pages.host_ptr(), pages.device_ptr(),
                           pages.memory_size(), hipMemcpyDeviceToHost,
                           stream));
  CUDA_TRY(hipStreamSynchronize(stream));
}

rmm::device_buffer reader::impl::decompress_page_data(
    const hostdevice_vector<gpu::ColumnChunkDesc> &chunks,
    const hostdevice_vector<gpu::PageInfo> &pages, hipStream_t stream) {
  auto for_each_codec_page = [&](parquet::Compression codec,
                                 const std::function<void(size_t)> &f) {
    for (size_t c = 0, page_count = 0; c < chunks.size(); c++) {
      const auto page_stride = chunks[c].max_num_pages;
      if (chunks[c].codec == codec) {
        for (int k = 0; k < page_stride; k++) {
          f(page_count + k);
        }
      }
      page_count += page_stride;
    }
  };

  // Brotli scratch memory for decompressing
  rmm::device_vector<uint8_t> debrotli_scratch;

  // Count the exact number of compressed pages
  size_t num_comp_pages = 0;
  size_t total_decomp_size = 0;
  std::array<std::pair<parquet::Compression, size_t>, 3> codecs{
      std::make_pair(parquet::GZIP, 0), std::make_pair(parquet::SNAPPY, 0),
      std::make_pair(parquet::BROTLI, 0)};

  for (auto &codec : codecs) {
    for_each_codec_page(codec.first, [&](size_t page) {
      total_decomp_size += pages[page].uncompressed_page_size;
      codec.second++;
      num_comp_pages++;
    });
    if (codec.first == parquet::BROTLI && codec.second > 0) {
      debrotli_scratch.resize(get_gpu_debrotli_scratch_size(codec.second));
    }
  }

  // Dispatch batches of pages to decompress for each codec
  rmm::device_buffer decomp_pages(total_decomp_size, stream);
  hostdevice_vector<gpu_inflate_input_s> inflate_in(0, num_comp_pages, stream);
  hostdevice_vector<gpu_inflate_status_s> inflate_out(0, num_comp_pages,
                                                      stream);

  size_t decomp_offset = 0;
  int32_t argc = 0;
  for (const auto &codec : codecs) {
    if (codec.second > 0) {
      int32_t start_pos = argc;

      for_each_codec_page(codec.first, [&](size_t page) {
        auto dst_base = static_cast<uint8_t *>(decomp_pages.data());
        inflate_in[argc].srcDevice = pages[page].page_data;
        inflate_in[argc].srcSize = pages[page].compressed_page_size;
        inflate_in[argc].dstDevice = dst_base + decomp_offset;
        inflate_in[argc].dstSize = pages[page].uncompressed_page_size;

        inflate_out[argc].bytes_written = 0;
        inflate_out[argc].status = static_cast<uint32_t>(-1000);
        inflate_out[argc].reserved = 0;

        pages[page].page_data = (uint8_t *)inflate_in[argc].dstDevice;
        decomp_offset += inflate_in[argc].dstSize;
        argc++;
      });

      CUDA_TRY(hipMemcpyAsync(
          inflate_in.device_ptr(start_pos), inflate_in.host_ptr(start_pos),
          sizeof(decltype(inflate_in)::value_type) * (argc - start_pos),
          hipMemcpyHostToDevice, stream));
      CUDA_TRY(hipMemcpyAsync(
          inflate_out.device_ptr(start_pos), inflate_out.host_ptr(start_pos),
          sizeof(decltype(inflate_out)::value_type) * (argc - start_pos),
          hipMemcpyHostToDevice, stream));
      switch (codec.first) {
        case parquet::GZIP:
          CUDA_TRY(gpuinflate(inflate_in.device_ptr(start_pos),
                              inflate_out.device_ptr(start_pos),
                              argc - start_pos, 1, stream))
          break;
        case parquet::SNAPPY:
          CUDA_TRY(gpu_unsnap(inflate_in.device_ptr(start_pos),
                              inflate_out.device_ptr(start_pos),
                              argc - start_pos, stream));
          break;
        case parquet::BROTLI:
          CUDA_TRY(gpu_debrotli(
              inflate_in.device_ptr(start_pos),
              inflate_out.device_ptr(start_pos), debrotli_scratch.data().get(),
              debrotli_scratch.size(), argc - start_pos, stream));
          break;
        default:
          CUDF_EXPECTS(false, "Unexpected decompression dispatch");
          break;
      }
      CUDA_TRY(hipMemcpyAsync(
          inflate_out.host_ptr(start_pos), inflate_out.device_ptr(start_pos),
          sizeof(decltype(inflate_out)::value_type) * (argc - start_pos),
          hipMemcpyDeviceToHost, stream));
    }
  }
  CUDA_TRY(hipStreamSynchronize(stream));

  // Update the page information in device memory with the updated value of
  // page_data; it now points to the uncompressed data buffer
  CUDA_TRY(hipMemcpyAsync(pages.device_ptr(), pages.host_ptr(),
                           pages.memory_size(), hipMemcpyHostToDevice,
                           stream));

  return decomp_pages;
}

void reader::impl::decode_page_data(
    const hostdevice_vector<gpu::ColumnChunkDesc> &chunks,
    const hostdevice_vector<gpu::PageInfo> &pages, size_t min_row,
    size_t total_rows, const std::vector<int> &chunk_map,
    std::vector<column_buffer> &out_buffers, hipStream_t stream) {
  auto is_dict_chunk = [](const gpu::ColumnChunkDesc &chunk) {
    return (chunk.data_type & 0x7) == BYTE_ARRAY && chunk.num_dict_pages > 0;
  };

  // Count the number of string dictionary entries
  // NOTE: Assumes first page in the chunk is always the dictionary page
  size_t total_str_dict_indexes = 0;
  for (size_t c = 0, page_count = 0; c < chunks.size(); c++) {
    if (is_dict_chunk(chunks[c])) {
      total_str_dict_indexes += pages[page_count].num_values;
    }
    page_count += chunks[c].max_num_pages;
  }

  // Build index for string dictionaries since they can't be indexed
  // directly due to variable-sized elements
  rmm::device_vector<gpu::nvstrdesc_s> str_dict_index;
  if (total_str_dict_indexes > 0) {
    str_dict_index.resize(total_str_dict_indexes);
  }

  // Update chunks with pointers to column data
  for (size_t c = 0, page_count = 0, str_ofs = 0; c < chunks.size(); c++) {
    if (is_dict_chunk(chunks[c])) {
      chunks[c].str_dict_index = str_dict_index.data().get() + str_ofs;
      str_ofs += pages[page_count].num_values;
    }
    chunks[c].column_data_base = out_buffers[chunk_map[c]].data();
    chunks[c].valid_map_base = out_buffers[chunk_map[c]].null_mask();
    page_count += chunks[c].max_num_pages;
  }

  CUDA_TRY(hipMemcpyAsync(chunks.device_ptr(), chunks.host_ptr(),
                           chunks.memory_size(), hipMemcpyHostToDevice,
                           stream));
  if (total_str_dict_indexes > 0) {
    CUDA_TRY(gpu::BuildStringDictionaryIndex(chunks.device_ptr(), chunks.size(),
                                             stream));
  }
  CUDA_TRY(gpu::DecodePageData(pages.device_ptr(), pages.size(),
                               chunks.device_ptr(), chunks.size(), total_rows,
                               min_row, stream));
  CUDA_TRY(hipMemcpyAsync(pages.host_ptr(), pages.device_ptr(),
                           pages.memory_size(), hipMemcpyDeviceToHost,
                           stream));
  CUDA_TRY(hipStreamSynchronize(stream));

  for (size_t i = 0; i < pages.size(); i++) {
    if (pages[i].num_rows > 0) {
      const size_t c = pages[i].chunk_idx;
      if (c < chunks.size()) {
        out_buffers[chunk_map[c]].null_count() +=
            pages[i].num_rows - pages[i].valid_count;
      }
    }
  }
}

reader::impl::impl(std::unique_ptr<datasource> source,
                   reader_options const &options,
                   rmm::mr::device_memory_resource *mr)
    : _source(std::move(source)), _mr(mr) {
  // Open and parse the source dataset metadata
  _metadata = std::make_unique<metadata>(_source.get());

  // Select only columns required by the options
  _selected_columns = _metadata->select_columns(
      options.columns, options.use_pandas_metadata, _pandas_index);

  // Store the index column (PANDAS-specific)
  _pandas_index = _metadata->get_pandas_index_name();

  // Override output timestamp resolution if requested
  if (options.timestamp_type.id() != EMPTY) {
    _timestamp_type = options.timestamp_type;
  }

  // Strings may be returned as either string or categorical columns
  _strings_to_categorical = options.strings_to_categorical;
}

table_with_metadata reader::impl::read(int skip_rows, int num_rows, int row_group,
                                       hipStream_t stream) {
  std::vector<std::unique_ptr<column>> out_columns;
  table_metadata out_metadata;

  // Select only row groups required
  const auto selected_row_groups =
      _metadata->select_row_groups(row_group, skip_rows, num_rows);

  if (selected_row_groups.size() != 0 && _selected_columns.size() != 0) {
    // Get a list of column data types
    std::vector<data_type> column_types;
    for (const auto &col : _selected_columns) {
      auto &col_schema =
          _metadata->schema[_metadata->row_groups[selected_row_groups[0].first]
                                .columns[col.first]
                                .schema_idx];
      auto col_type = to_type_id(col_schema.type, col_schema.converted_type,
                                 _strings_to_categorical, _timestamp_type.id(),
                                 col_schema.decimal_scale);
      CUDF_EXPECTS(col_type != type_id::EMPTY, "Unknown type");
      column_types.emplace_back(col_type);
    }

    // Descriptors for all the chunks that make up the selected columns
    const auto num_columns = _selected_columns.size();
    const auto num_chunks = selected_row_groups.size() * num_columns;
    hostdevice_vector<gpu::ColumnChunkDesc> chunks(0, num_chunks, stream);

    // Association between each column chunk and its column
    std::vector<int> chunk_map(num_chunks);

    // Tracker for eventually deallocating compressed and uncompressed data
    std::vector<rmm::device_buffer> page_data(num_chunks);

    // Initialize column chunk information
    size_t total_decompressed_size = 0;
    auto remaining_rows = num_rows;
    for (const auto &rg : selected_row_groups) {
      auto row_group = _metadata->row_groups[rg.first];
      auto row_group_start = rg.second;
      auto row_group_rows = std::min<int>(remaining_rows, row_group.num_rows);

      for (size_t i = 0; i < num_columns; ++i) {
        auto col = _selected_columns[i];
        auto &col_meta = row_group.columns[col.first].meta_data;
        auto &col_schema =
            _metadata->schema[row_group.columns[col.first].schema_idx];

        // Spec requires each row group to contain exactly one chunk for every
        // column. If there are too many or too few, continue with best effort
        if (col.second != _metadata->get_column_name(col_meta.path_in_schema)) {
          std::cerr << "Detected mismatched column chunk" << std::endl;
          continue;
        }
        if (chunks.size() >= chunks.max_size()) {
          std::cerr << "Detected too many column chunks" << std::endl;
          continue;
        }

        int32_t type_width;
        int32_t clock_rate;
        int8_t converted_type;
        std::tie(type_width, clock_rate, converted_type) = conversion_info(
            column_types[i].id(), _timestamp_type.id(), col_schema.type,
            col_schema.converted_type, col_schema.type_length);

        uint8_t *d_compdata = nullptr;
        if (col_meta.total_compressed_size != 0) {
          const auto offset = (col_meta.dictionary_page_offset != 0)
                                  ? std::min(col_meta.data_page_offset,
                                             col_meta.dictionary_page_offset)
                                  : col_meta.data_page_offset;
          auto buffer =
              _source->get_buffer(offset, col_meta.total_compressed_size);
          page_data[chunks.size()] = rmm::device_buffer(buffer->data(), buffer->size(), stream);
          d_compdata = static_cast<uint8_t *>(page_data[chunks.size()].data());
        }
        chunks.insert(gpu::ColumnChunkDesc(
            col_meta.total_compressed_size, d_compdata, col_meta.num_values,
            col_schema.type, type_width, row_group_start, row_group_rows,
            col_schema.max_definition_level, col_schema.max_repetition_level,
            required_bits(col_schema.max_definition_level),
            required_bits(col_schema.max_repetition_level), col_meta.codec,
            converted_type, col_schema.decimal_scale, clock_rate));

        // Map each column chunk to its column index
        chunk_map[chunks.size() - 1] = i;

        if (col_meta.codec != Compression::UNCOMPRESSED) {
          total_decompressed_size += col_meta.total_uncompressed_size;
        }
      }
      remaining_rows -= row_group.num_rows;
    }
    assert(remaining_rows <= 0);

    // Process dataset chunk pages into output columns
    const auto total_pages = count_page_headers(chunks, stream);
    if (total_pages > 0) {
      hostdevice_vector<gpu::PageInfo> pages(total_pages, total_pages, stream);
      rmm::device_buffer decomp_page_data;

      decode_page_headers(chunks, pages, stream);
      if (total_decompressed_size > 0) {
        decomp_page_data = decompress_page_data(chunks, pages, stream);
        // Free compressed data
        for (size_t c = 0; c < chunks.size(); c++) {
          if (chunks[c].codec != parquet::Compression::UNCOMPRESSED) {
            page_data[c].resize(0);
            page_data[c].shrink_to_fit();
          }
        }
      }

      std::vector<column_buffer> out_buffers;
      for (size_t i = 0; i < column_types.size(); ++i) {
        out_buffers.emplace_back(column_types[i], num_rows, stream, _mr);
      }

      decode_page_data(chunks, pages, skip_rows, num_rows, chunk_map,
                       out_buffers, stream);

      for (size_t i = 0; i < column_types.size(); ++i) {
        out_columns.emplace_back(make_column(column_types[i], num_rows,
                                             out_buffers[i], stream, _mr));
      }
    }
  }

  // Return column names (must match order of returned columns)
  out_metadata.column_names.resize(_selected_columns.size());
  for (size_t i = 0; i < _selected_columns.size(); i++) {
    out_metadata.column_names[i] = _selected_columns[i].second;
  }
  // Return user metadata
  for (const auto& kv : _metadata->key_value_metadata) {
    out_metadata.user_data.insert({kv.key, kv.value});
  }

  return { std::make_unique<table>(std::move(out_columns)), std::move(out_metadata) };
}

// Forward to implementation
reader::reader(std::string filepath, reader_options const &options,
               rmm::mr::device_memory_resource *mr)
    : _impl(std::make_unique<impl>(datasource::create(filepath), options, mr)) {
}

// Forward to implementation
reader::reader(const char *buffer, size_t length, reader_options const &options,
               rmm::mr::device_memory_resource *mr)
    : _impl(std::make_unique<impl>(datasource::create(buffer, length), options,
                                   mr)) {}

// Forward to implementation
reader::reader(std::shared_ptr<arrow::io::RandomAccessFile> file,
               reader_options const &options,
               rmm::mr::device_memory_resource *mr)
    : _impl(std::make_unique<impl>(datasource::create(file), options, mr)) {}

// Destructor within this translation unit
reader::~reader() = default;

// Forward to implementation
std::string reader::get_pandas_index() { return _impl->get_pandas_index(); }

// Forward to implementation
table_with_metadata reader::read_all(hipStream_t stream) {
  return _impl->read(0, -1, -1, stream);
}

// Forward to implementation
table_with_metadata reader::read_row_group(size_type row_group,
                                           hipStream_t stream) {
  return _impl->read(0, -1, row_group, stream);
}

// Forward to implementation
table_with_metadata reader::read_rows(size_type skip_rows,
                                      size_type num_rows,
                                      hipStream_t stream) {
  return _impl->read(skip_rows, (num_rows != 0) ? num_rows : -1, -1, stream);
}

}  // namespace parquet
}  // namespace detail
}  // namespace io
}  // namespace experimental
}  // namespace cudf
