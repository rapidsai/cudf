#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019-2020, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

/**
 * @file reader_impl.cu
 * @brief cuDF-IO Parquet reader class implementation
 */

#include "reader_impl.hpp"

#include <io/comp/gpuinflate.h>

#include <cudf/table/table.hpp>
#include <cudf/utilities/error.hpp>
#include <cudf/utilities/traits.hpp>

#include <rmm/thrust_rmm_allocator.h>
#include <rmm/device_buffer.hpp>
#include <rmm/device_uvector.hpp>

#include <algorithm>
#include <array>
#include <numeric>
#include <regex>

namespace cudf {
namespace io {
namespace detail {
namespace parquet {
// Import functionality that's independent of legacy code
using namespace cudf::io::parquet;
using namespace cudf::io;

// bit space we are reserving in column_buffer::user_data
constexpr uint32_t PARQUET_COLUMN_BUFFER_SCHEMA_MASK          = (0xffffff);
constexpr uint32_t PARQUET_COLUMN_BUFFER_FLAG_LIST_TERMINATED = (1 << 24);

namespace {
/**
 * @brief Function that translates Parquet datatype to cuDF type enum
 */
type_id to_type_id(SchemaElement const &schema,
                   bool strings_to_categorical,
                   type_id timestamp_type_id)
{
  parquet::Type physical         = schema.type;
  parquet::ConvertedType logical = schema.converted_type;
  int32_t decimal_scale          = schema.decimal_scale;

  // Logical type used for actual data interpretation; the legacy converted type
  // is superceded by 'logical' type whenever available.
  switch (logical) {
    case parquet::UINT_8: return type_id::UINT8;
    case parquet::INT_8: return type_id::INT8;
    case parquet::UINT_16: return type_id::UINT16;
    case parquet::INT_16: return type_id::INT16;
    case parquet::UINT_32: return type_id::UINT32;
    case parquet::UINT_64: return type_id::UINT64;
    case parquet::DATE: return type_id::TIMESTAMP_DAYS;
    case parquet::TIME_MILLIS:
      return (timestamp_type_id != type_id::EMPTY) ? timestamp_type_id
                                                   : type_id::DURATION_MILLISECONDS;
    case parquet::TIME_MICROS:
      return (timestamp_type_id != type_id::EMPTY) ? timestamp_type_id
                                                   : type_id::DURATION_MICROSECONDS;
    case parquet::TIMESTAMP_MICROS:
      return (timestamp_type_id != type_id::EMPTY) ? timestamp_type_id
                                                   : type_id::TIMESTAMP_MICROSECONDS;
    case parquet::TIMESTAMP_MILLIS:
      return (timestamp_type_id != type_id::EMPTY) ? timestamp_type_id
                                                   : type_id::TIMESTAMP_MILLISECONDS;
    case parquet::DECIMAL:
      if (decimal_scale != 0 || (physical != parquet::INT32 && physical != parquet::INT64)) {
        return type_id::FLOAT64;
      }
      break;

    // maps are just List<Struct<>>.
    case parquet::MAP:
    case parquet::LIST: return type_id::LIST;

    default: break;
  }

  // is it simply a struct?
  if (schema.is_struct()) { return type_id::STRUCT; }

  // Physical storage type supported by Parquet; controls the on-disk storage
  // format in combination with the encoding type.
  switch (physical) {
    case parquet::BOOLEAN: return type_id::BOOL8;
    case parquet::INT32: return type_id::INT32;
    case parquet::INT64: return type_id::INT64;
    case parquet::FLOAT: return type_id::FLOAT32;
    case parquet::DOUBLE: return type_id::FLOAT64;
    case parquet::BYTE_ARRAY:
    case parquet::FIXED_LEN_BYTE_ARRAY:
      // Can be mapped to INT32 (32-bit hash) or STRING
      return strings_to_categorical ? type_id::INT32 : type_id::STRING;
    case parquet::INT96:
      return (timestamp_type_id != type_id::EMPTY) ? timestamp_type_id
                                                   : type_id::TIMESTAMP_NANOSECONDS;
    default: break;
  }

  return type_id::EMPTY;
}

/**
 * @brief Function that translates cuDF time unit to Parquet clock frequency
 */
constexpr int32_t to_clockrate(type_id timestamp_type_id)
{
  switch (timestamp_type_id) {
    case type_id::DURATION_SECONDS: return 1;
    case type_id::DURATION_MILLISECONDS: return 1000;
    case type_id::DURATION_MICROSECONDS: return 1000000;
    case type_id::DURATION_NANOSECONDS: return 1000000000;
    case type_id::TIMESTAMP_SECONDS: return 1;
    case type_id::TIMESTAMP_MILLISECONDS: return 1000;
    case type_id::TIMESTAMP_MICROSECONDS: return 1000000;
    case type_id::TIMESTAMP_NANOSECONDS: return 1000000000;
    default: return 0;
  }
}

/**
 * @brief Function that returns the required the number of bits to store a value
 */
template <typename T = uint8_t>
T required_bits(uint32_t max_level)
{
  return static_cast<T>(CompactProtocolReader::NumRequiredBits(max_level));
}

std::tuple<int32_t, int32_t, int8_t> conversion_info(type_id column_type_id,
                                                     type_id timestamp_type_id,
                                                     parquet::Type physical,
                                                     int8_t converted,
                                                     int32_t length)
{
  int32_t type_width = (physical == parquet::FIXED_LEN_BYTE_ARRAY) ? length : 0;
  int32_t clock_rate = 0;
  if (column_type_id == type_id::INT8 or column_type_id == type_id::UINT8) {
    type_width = 1;  // I32 -> I8
  } else if (column_type_id == type_id::INT16 or column_type_id == type_id::UINT16) {
    type_width = 2;  // I32 -> I16
  } else if (column_type_id == type_id::INT32) {
    type_width = 4;  // str -> hash32
  } else if (is_chrono(data_type{column_type_id})) {
    clock_rate = to_clockrate(timestamp_type_id);
  }

  int8_t converted_type = converted;
  if (converted_type == parquet::DECIMAL && column_type_id != type_id::FLOAT64) {
    converted_type = parquet::UNKNOWN;  // Not converting to float64
  }
  return std::make_tuple(type_width, clock_rate, converted_type);
}

}  // namespace

std::string name_from_path(const std::vector<std::string> &path_in_schema)
{
  // For the case of lists, we will see a schema that looks like:
  // a.list.element.list.element
  // where each (list.item) pair represents a level of nesting.  According to the parquet spec,
  // https://github.com/apache/parquet-format/blob/master/LogicalTypes.md
  // the initial field must be named "list" and the inner element must be named "element".
  // If we are dealing with a list, we want to return the topmost name of the group ("a").
  //
  // For other nested schemas, like structs we just want to return the bottom-most name. For
  // example a struct with the schema
  // b.employee.id,  the column representing "id" should simply be named "id".
  //
  // In short, this means : return the highest level of the schema that does not have list
  // definitions underneath it.
  //
  std::string s = (path_in_schema.size() > 0) ? path_in_schema[0] : "";
  for (size_t i = 1; i < path_in_schema.size(); i++) {
    // The Parquet spec requires that the outer schema field is named "list". However it also
    // provides a list of backwards compatibility cases that are applicable as well.  Currently
    // we are only handling the formal spec.  This will get cleaned up and improved when we add
    // support for structs. The correct thing to do will probably be to examine the type of
    // the SchemaElement itself to concretely identify the start of a nested type of any kind rather
    // than trying to derive it from the path string.
    if (path_in_schema[i] == "list") {
      // Again, strictly speaking, the Parquet spec says the inner field should be named
      // "element", but there are some backwards compatibility issues that we have seen in the
      // wild. For example, Pandas calls the field "item".  We will allow any name for now.
      i++;
      continue;
    }
    // otherwise, we've got a real nested column. update the name
    s = path_in_schema[i];
  }
  return s;
}

/**
 * @brief Class for parsing dataset metadata
 */
struct metadata : public FileMetaData {
  explicit metadata(datasource *source)
  {
    constexpr auto header_len = sizeof(file_header_s);
    constexpr auto ender_len  = sizeof(file_ender_s);

    const auto len           = source->size();
    const auto header_buffer = source->host_read(0, header_len);
    const auto header        = reinterpret_cast<const file_header_s *>(header_buffer->data());
    const auto ender_buffer  = source->host_read(len - ender_len, ender_len);
    const auto ender         = reinterpret_cast<const file_ender_s *>(ender_buffer->data());
    CUDF_EXPECTS(len > header_len + ender_len, "Incorrect data source");
    CUDF_EXPECTS(header->magic == parquet_magic && ender->magic == parquet_magic,
                 "Corrupted header or footer");
    CUDF_EXPECTS(ender->footer_len != 0 && ender->footer_len <= (len - header_len - ender_len),
                 "Incorrect footer length");

    const auto buffer = source->host_read(len - ender->footer_len - ender_len, ender->footer_len);
    CompactProtocolReader cp(buffer->data(), ender->footer_len);
    CUDF_EXPECTS(cp.read(this), "Cannot parse metadata");
    CUDF_EXPECTS(cp.InitSchema(this), "Cannot initialize schema");
  }
};

class aggregate_metadata {
  std::vector<metadata> const per_file_metadata;
  std::map<std::string, std::string> const agg_keyval_map;
  size_type const num_rows;
  size_type const num_row_groups;
  /**
   * @brief Create a metadata object from each element in the source vector
   */
  auto metadatas_from_sources(std::vector<std::unique_ptr<datasource>> const &sources)
  {
    std::vector<metadata> metadatas;
    std::transform(
      sources.cbegin(), sources.cend(), std::back_inserter(metadatas), [](auto const &source) {
        return metadata(source.get());
      });
    return metadatas;
  }

  /**
   * @brief Merge the keyvalue maps from each per-file metadata object into a single map.
   */
  auto merge_keyval_metadata()
  {
    std::map<std::string, std::string> merged;
    // merge key/value maps TODO: warn/throw if there are mismatches?
    for (auto const &pfm : per_file_metadata) {
      for (auto const &kv : pfm.key_value_metadata) { merged[kv.key] = kv.value; }
    }
    return merged;
  }

  /**
   * @brief Sums up the number of rows of each source
   */
  size_type calc_num_rows() const
  {
    return std::accumulate(
      per_file_metadata.begin(), per_file_metadata.end(), 0, [](auto &sum, auto &pfm) {
        return sum + pfm.num_rows;
      });
  }

  /**
   * @brief Sums up the number of row groups of each source
   */
  size_type calc_num_row_groups() const
  {
    return std::accumulate(
      per_file_metadata.begin(), per_file_metadata.end(), 0, [](auto &sum, auto &pfm) {
        return sum + pfm.row_groups.size();
      });
  }

 public:
  aggregate_metadata(std::vector<std::unique_ptr<datasource>> const &sources)
    : per_file_metadata(metadatas_from_sources(sources)),
      agg_keyval_map(merge_keyval_metadata()),
      num_rows(calc_num_rows()),
      num_row_groups(calc_num_row_groups())
  {
    // Verify that the input files have matching numbers of columns
    size_type num_cols = -1;
    for (auto const &pfm : per_file_metadata) {
      if (pfm.row_groups.size() != 0) {
        if (num_cols == -1)
          num_cols = pfm.row_groups[0].columns.size();
        else
          CUDF_EXPECTS(num_cols == static_cast<size_type>(pfm.row_groups[0].columns.size()),
                       "All sources must have the same number of columns");
      }
    }
    // Verify that the input files have matching schemas
    for (auto const &pfm : per_file_metadata) {
      CUDF_EXPECTS(per_file_metadata[0].schema == pfm.schema,
                   "All sources must have the same schemas");
    }
  }

  auto const &get_row_group(size_type row_group_index, size_type src_idx) const
  {
    CUDF_EXPECTS(src_idx >= 0 && src_idx < static_cast<size_type>(per_file_metadata.size()),
                 "invalid source index");
    return per_file_metadata[src_idx].row_groups[row_group_index];
  }

  auto const &get_column_metadata(size_type row_group_index,
                                  size_type src_idx,
                                  int schema_idx) const
  {
    auto col = std::find_if(
      per_file_metadata[src_idx].row_groups[row_group_index].columns.begin(),
      per_file_metadata[src_idx].row_groups[row_group_index].columns.end(),
      [schema_idx](ColumnChunk const &col) { return col.schema_idx == schema_idx ? true : false; });
    CUDF_EXPECTS(col != std::end(per_file_metadata[src_idx].row_groups[row_group_index].columns),
                 "Found no metadata for schema index");
    return col->meta_data;
  }

  auto get_num_rows() const { return num_rows; }

  auto get_num_row_groups() const { return num_row_groups; }

  auto const &get_schema(int schema_idx) const { return per_file_metadata[0].schema[schema_idx]; }

  auto const &get_key_value_metadata() const { return agg_keyval_map; }

  /**
   * @brief Gets the concrete nesting depth of output cudf columns
   *
   * @param schema_index Schema index of the input column
   *
   * @return comma-separated index column names in quotes
   */
  inline int get_output_nesting_depth(int schema_index) const
  {
    auto &pfm = per_file_metadata[0];
    int depth = 0;

    // walk upwards, skipping repeated fields
    while (schema_index > 0) {
      if (!pfm.schema[schema_index].is_stub()) { depth++; }
      schema_index = pfm.schema[schema_index].parent_idx;
    }
    return depth;
  }

  /**
   * @brief Extracts the pandas "index_columns" section
   *
   * PANDAS adds its own metadata to the key_value section when writing out the
   * dataframe to a file to aid in exact reconstruction. The JSON-formatted
   * metadata contains the index column(s) and PANDA-specific datatypes.
   *
   * @return comma-separated index column names in quotes
   */
  std::string get_pandas_index() const
  {
    auto it = agg_keyval_map.find("pandas");
    if (it != agg_keyval_map.end()) {
      // Captures a list of quoted strings found inside square brackets after `"index_columns":`
      // Inside quotes supports newlines, brackets, escaped quotes, etc.
      // One-liner regex:
      // "index_columns"\s*:\s*\[\s*((?:"(?:|(?:.*?(?![^\\]")).?)[^\\]?",?\s*)*)\]
      // Documented below.
      std::regex index_columns_expr{
        R"("index_columns"\s*:\s*\[\s*)"  // match preamble, opening square bracket, whitespace
        R"(()"                            // Open first capturing group
        R"((?:")"                         // Open non-capturing group match opening quote
        R"((?:|(?:.*?(?![^\\]")).?))"     // match empty string or anything between quotes
        R"([^\\]?")"                      // Match closing non-escaped quote
        R"(,?\s*)"                        // Match optional comma and whitespace
        R"()*)"                           // Close non-capturing group and repeat 0 or more times
        R"())"                            // Close first capturing group
        R"(\])"                           // Match closing square brackets
      };
      std::smatch sm;
      if (std::regex_search(it->second, sm, index_columns_expr)) { return std::move(sm[1].str()); }
    }
    return "";
  }

  /**
   * @brief Extracts the column name(s) used for the row indexes in a dataframe
   *
   * @param names List of column names to load, where index column name(s) will be added
   */
  void add_pandas_index_names(std::vector<std::string> &names) const
  {
    auto str = get_pandas_index();
    if (str.length() != 0) {
      std::regex index_name_expr{R"(\"((?:\\.|[^\"])*)\")"};
      std::smatch sm;
      while (std::regex_search(str, sm, index_name_expr)) {
        if (sm.size() == 2) {  // 2 = whole match, first item
          if (std::find(names.begin(), names.end(), sm[1].str()) == names.end()) {
            std::regex esc_quote{R"(\\")"};
            names.emplace_back(std::move(std::regex_replace(sm[1].str(), esc_quote, R"(")")));
          }
        }
        str = sm.suffix();
      }
    }
  }

  struct row_group_info {
    size_type const index;
    size_t const start_row;  // TODO source index
    size_type const source_index;
    row_group_info(size_type index, size_t start_row, size_type source_index)
      : index(index), start_row(start_row), source_index(source_index)
    {
    }
  };

  /**
   * @brief Filters and reduces down to a selection of row groups
   *
   * @param row_groups Lists of row group to reads, one per source
   * @param row_start Starting row of the selection
   * @param row_count Total number of rows selected
   *
   * @return List of row group indexes and its starting row
   */
  auto select_row_groups(std::vector<std::vector<size_type>> const &row_groups,
                         size_type &row_start,
                         size_type &row_count) const
  {
    if (!row_groups.empty()) {
      std::vector<row_group_info> selection;
      CUDF_EXPECTS(row_groups.size() == per_file_metadata.size(),
                   "Must specify row groups for each source");

      row_count = 0;
      for (size_t src_idx = 0; src_idx < row_groups.size(); ++src_idx) {
        for (auto const &rowgroup_idx : row_groups[src_idx]) {
          CUDF_EXPECTS(
            rowgroup_idx >= 0 &&
              rowgroup_idx < static_cast<size_type>(per_file_metadata[src_idx].row_groups.size()),
            "Invalid rowgroup index");
          selection.emplace_back(rowgroup_idx, row_count, src_idx);
          row_count += get_row_group(rowgroup_idx, src_idx).num_rows;
        }
      }
      return selection;
    }

    row_start = std::max(row_start, 0);
    if (row_count < 0) {
      row_count = static_cast<size_type>(
        std::min<int64_t>(get_num_rows(), std::numeric_limits<size_type>::max()));
    }
    row_count = min(row_count, get_num_rows() - row_start);
    CUDF_EXPECTS(row_count >= 0, "Invalid row count");
    CUDF_EXPECTS(row_start <= get_num_rows(), "Invalid row start");

    std::vector<row_group_info> selection;
    size_type count = 0;
    for (size_t src_idx = 0; src_idx < per_file_metadata.size(); ++src_idx) {
      for (size_t rg_idx = 0; rg_idx < per_file_metadata[src_idx].row_groups.size(); ++rg_idx) {
        auto const chunk_start_row = count;
        count += get_row_group(rg_idx, src_idx).num_rows;
        if (count > row_start || count == 0) {
          selection.emplace_back(rg_idx, chunk_start_row, src_idx);
        }
        if (count >= row_start + row_count) { break; }
      }
    }

    return selection;
  }

  /**
   * @brief Build input and output column structures based on schema input. Recursive.
   *
   * @param[in,out] schema_idx Schema index to build information for. This value gets
   * incremented as the function recurses.
   * @param[out] input_columns Input column information (source data in the file)
   * @param[out] output_columns Output column structure (resulting cudf columns)
   * @param[in,out] nesting A stack keeping track of child column indices so we can
   * reproduce the linear list of output columns that correspond to an input column.
   * @param[in] strings_to_categorical Type conversion parameter
   * @param[in] timestamp_type_id Type conversion parameter
   *
   */
  void build_column_info(int &schema_idx,
                         std::vector<input_column_info> &input_columns,
                         std::vector<column_buffer> &output_columns,
                         std::deque<int> &nesting,
                         bool strings_to_categorical,
                         type_id timestamp_type_id) const
  {
    int start_schema_idx = schema_idx;
    auto const &schema   = get_schema(schema_idx);
    schema_idx++;

    // if I am a stub, continue on
    if (schema.is_stub()) {
      // is this legit?
      CUDF_EXPECTS(schema.num_children == 1, "Unexpected number of children for stub");
      build_column_info(schema_idx,
                        input_columns,
                        output_columns,
                        nesting,
                        strings_to_categorical,
                        timestamp_type_id);
      return;
    }

    // if we're at the root, this is a new output column
    int index = (int)output_columns.size();
    nesting.push_back(static_cast<int>(output_columns.size()));
    output_columns.emplace_back(
      data_type{to_type_id(schema, strings_to_categorical, timestamp_type_id)},
      schema.repetition_type == OPTIONAL ? true : false);
    column_buffer &output_col = output_columns.back();
    output_col.name           = schema.name;

    // build each child
    for (int idx = 0; idx < schema.num_children; idx++) {
      build_column_info(schema_idx,
                        input_columns,
                        output_col.children,
                        nesting,
                        strings_to_categorical,
                        timestamp_type_id);
    }

    // if I have no children, we're at a leaf and I'm an input column (that is, one with actual
    // data stored) so add me to the list.
    if (schema.num_children == 0) {
      input_columns.emplace_back(input_column_info{start_schema_idx, schema.name});
      input_column_info &input_col = input_columns.back();
      std::copy(nesting.begin(), nesting.end(), std::back_inserter(input_col.nesting));
    }

    nesting.pop_back();
  }

  /**
   * @brief Filters and reduces down to a selection of columns
   *
   * @param use_names List of column names to select
   * @param include_index Whether to always include the PANDAS index column(s)
   * @param strings_to_categorical Type conversion parameter
   * @param timestamp_type_id Type conversion parameter
   *
   * @return input column information, output column information, list of output column schema
   * indices
   */
  auto select_columns(std::vector<std::string> const &use_names,
                      bool include_index,
                      bool strings_to_categorical,
                      type_id timestamp_type_id) const
  {
    auto const &pfm = per_file_metadata[0];

    // determine the list of output columns
    //
    // there is not necessarily a 1:1 mapping between input columns and output columns.
    // For example, parquet does not explicitly store a ColumnChunkDesc for struct columns.
    // The "structiness" is simply implied by the schema.  For example, this schema:
    //  required group field_id=1 name {
    //    required binary field_id=2 firstname (String);
    //    required binary field_id=3 middlename (String);
    //    required binary field_id=4 lastname (String);
    // }
    // will only contain 3 internal columns of data (firstname, middlename, lastname).  But of
    // course "name" is ultimately the struct column we want to return.
    //
    // "firstname", "middlename" and "lastname" represent the input columns in the file that we
    // process to produce the final cudf "name" column.
    //
    std::vector<int> output_column_schemas;
    if (use_names.empty()) {
      // walk the schema and choose all top level columns
      for (size_t schema_idx = 1; schema_idx < pfm.schema.size(); schema_idx++) {
        auto const &schema = pfm.schema[schema_idx];
        if (schema.parent_idx == 0) { output_column_schemas.push_back(schema_idx); }
      }
    } else {
      // Load subset of columns; include PANDAS index unless excluded
      std::vector<std::string> local_use_names = use_names;
      if (include_index) { add_pandas_index_names(local_use_names); }
      for (const auto &use_name : local_use_names) {
        for (size_t schema_idx = 1; schema_idx < pfm.schema.size(); schema_idx++) {
          auto const &schema = pfm.schema[schema_idx];
          if (use_name == schema.name) { output_column_schemas.push_back(schema_idx); }
        }
      }
    }

    // construct input and output output column info
    std::vector<column_buffer> output_columns;
    output_columns.reserve(output_column_schemas.size());
    std::vector<input_column_info> input_columns;
    std::deque<int> nesting;
    for (size_t idx = 0; idx < output_column_schemas.size(); idx++) {
      int schema_index = output_column_schemas[idx];
      build_column_info(schema_index,
                        input_columns,
                        output_columns,
                        nesting,
                        strings_to_categorical,
                        timestamp_type_id);
    }

    return std::make_tuple(
      std::move(input_columns), std::move(output_columns), std::move(output_column_schemas));
  }
};

/**
 * @brief Generate depth remappings for repetition and definition levels.
 *
 * When dealing with columns that contain lists, we must examine incoming
 * repetition and definition level pairs to determine what range of output nesting
 * is indicated when adding new values.  This function generates the mappings of
 * the R/D levels to those start/end bounds
 *
 * @param remap Maps column schema index to the R/D remapping vectors for that column
 * @param src_col_schema The column schema to generate the new mapping for
 * @param md File metadata information
 *
 */
void generate_depth_remappings(std::map<int, std::pair<std::vector<int>, std::vector<int>>> &remap,
                               int src_col_schema,
                               aggregate_metadata const &md)
{
  // already generated for this level
  if (remap.find(src_col_schema) != remap.end()) { return; }
  auto schema   = md.get_schema(src_col_schema);
  int max_depth = md.get_output_nesting_depth(src_col_schema);

  CUDF_EXPECTS(remap.find(src_col_schema) == remap.end(),
               "Attempting to remap a schema more than once");
  auto inserted =
    remap.insert(std::pair<int, std::pair<std::vector<int>, std::vector<int>>>{src_col_schema, {}});
  auto &depth_remap = inserted.first->second;

  std::vector<int> &rep_depth_remap = (depth_remap.first);
  rep_depth_remap.resize(schema.max_repetition_level + 1);
  std::vector<int> &def_depth_remap = (depth_remap.second);
  def_depth_remap.resize(schema.max_definition_level + 1);

  // the key:
  // for incoming level values  R/D
  // add values starting at the shallowest nesting level X has repetition level R
  // until you reach the deepest nesting level Y that corresponds to the repetition level R1
  // held by the nesting level that has definition level D
  //
  // Example: a 3 level struct with a list at the bottom
  //
  //                     R / D   Depth
  // level0              0 / 1     0
  //   level1            0 / 2     1
  //     level2          0 / 3     2
  //       list          0 / 3     3
  //         element     1 / 4     4
  //
  // incoming R/D : 0, 0  -> add values from depth 0 to 3   (def level 0 always maps to depth 0)
  // incoming R/D : 0, 1  -> add values from depth 0 to 3
  // incoming R/D : 0, 2  -> add values from depth 0 to 3
  // incoming R/D : 1, 4  -> add values from depth 4 to 4
  //
  // Note : the -validity- of values is simply checked by comparing the incoming D value against the
  // D value of the given nesting level (incoming D >= the D for the nesting level == valid,
  // otherwise NULL).  The tricky part is determining what nesting levels to add values at.
  //
  // For schemas with no repetition level (no lists), X is always 0 and Y is always max nesting
  // depth.
  //

  // compute "X" from above
  for (int s_idx = schema.max_repetition_level; s_idx >= 0; s_idx--) {
    auto find_shallowest = [&](int r) {
      int shallowest = -1;
      int cur_depth  = max_depth - 1;
      int schema_idx = src_col_schema;
      while (schema_idx > 0) {
        auto cur_schema = md.get_schema(schema_idx);
        if (cur_schema.max_repetition_level == r) {
          // if this is a repeated field, map it one level deeper
          shallowest = cur_schema.is_stub() ? cur_depth + 1 : cur_depth;
        }
        if (!cur_schema.is_stub()) { cur_depth--; }
        schema_idx = cur_schema.parent_idx;
      }
      return shallowest;
    };
    rep_depth_remap[s_idx] = find_shallowest(s_idx);
  }

  // compute "Y" from above
  for (int s_idx = schema.max_definition_level; s_idx >= 0; s_idx--) {
    auto find_deepest = [&](int d) {
      SchemaElement prev_schema;
      int schema_idx = src_col_schema;
      int r1         = 0;
      while (schema_idx > 0) {
        SchemaElement cur_schema = md.get_schema(schema_idx);
        if (cur_schema.max_definition_level == d) {
          // if this is a repeated field, map it one level deeper
          r1 = cur_schema.is_stub() ? prev_schema.max_repetition_level
                                    : cur_schema.max_repetition_level;
          break;
        }
        prev_schema = cur_schema;
        schema_idx  = cur_schema.parent_idx;
      }

      // we now know R1 from above. return the deepest nesting level that has the
      // same repetition level
      schema_idx = src_col_schema;
      int depth  = max_depth - 1;
      while (schema_idx > 0) {
        SchemaElement cur_schema = md.get_schema(schema_idx);
        if (cur_schema.max_repetition_level == r1) {
          // if this is a repeated field, map it one level deeper
          depth = cur_schema.is_stub() ? depth + 1 : depth;
          break;
        }
        if (!cur_schema.is_stub()) { depth--; }
        prev_schema = cur_schema;
        schema_idx  = cur_schema.parent_idx;
      }
      return depth;
    };
    def_depth_remap[s_idx] = find_deepest(s_idx);
  }
}

/**
 * @copydoc cudf::io::detail::parquet::read_column_chunks
 */
void reader::impl::read_column_chunks(
  std::vector<rmm::device_buffer> &page_data,
  hostdevice_vector<gpu::ColumnChunkDesc> &chunks,  // TODO const?
  size_t begin_chunk,
  size_t end_chunk,
  const std::vector<size_t> &column_chunk_offsets,
  std::vector<size_type> const &chunk_source_map,
  hipStream_t stream)
{
  // Transfer chunk data, coalescing adjacent chunks
  for (size_t chunk = begin_chunk; chunk < end_chunk;) {
    const size_t io_offset   = column_chunk_offsets[chunk];
    size_t io_size           = chunks[chunk].compressed_size;
    size_t next_chunk        = chunk + 1;
    const bool is_compressed = (chunks[chunk].codec != parquet::Compression::UNCOMPRESSED);
    while (next_chunk < end_chunk) {
      const size_t next_offset = column_chunk_offsets[next_chunk];
      const bool is_next_compressed =
        (chunks[next_chunk].codec != parquet::Compression::UNCOMPRESSED);
      if (next_offset != io_offset + io_size || is_next_compressed != is_compressed) {
        // Can't merge if not contiguous or mixing compressed and uncompressed
        // Not coalescing uncompressed with compressed chunks is so that compressed buffers can be
        // freed earlier (immediately after decompression stage) to limit peak memory requirements
        break;
      }
      io_size += chunks[next_chunk].compressed_size;
      next_chunk++;
    }
    if (io_size != 0) {
      auto buffer         = _sources[chunk_source_map[chunk]]->host_read(io_offset, io_size);
      page_data[chunk]    = rmm::device_buffer(buffer->data(), buffer->size(), stream);
      uint8_t *d_compdata = static_cast<uint8_t *>(page_data[chunk].data());
      do {
        chunks[chunk].compressed_data = d_compdata;
        d_compdata += chunks[chunk].compressed_size;
      } while (++chunk != next_chunk);
    } else {
      chunk = next_chunk;
    }
  }
}

/**
 * @copydoc cudf::io::detail::parquet::count_page_headers
 */
size_t reader::impl::count_page_headers(hostdevice_vector<gpu::ColumnChunkDesc> &chunks,
                                        hipStream_t stream)
{
  size_t total_pages = 0;

  chunks.host_to_device(stream);
  CUDA_TRY(gpu::DecodePageHeaders(chunks.device_ptr(), chunks.size(), stream));
  chunks.device_to_host(stream, true);

  for (size_t c = 0; c < chunks.size(); c++) {
    total_pages += chunks[c].num_data_pages + chunks[c].num_dict_pages;
  }

  return total_pages;
}

/**
 * @copydoc cudf::io::detail::parquet::decode_page_headers
 */
void reader::impl::decode_page_headers(hostdevice_vector<gpu::ColumnChunkDesc> &chunks,
                                       hostdevice_vector<gpu::PageInfo> &pages,
                                       hipStream_t stream)
{
  // IMPORTANT : if you change how pages are stored within a chunk (dist pages, then data pages),
  // please update preprocess_nested_columns to reflect this.
  for (size_t c = 0, page_count = 0; c < chunks.size(); c++) {
    chunks[c].max_num_pages = chunks[c].num_data_pages + chunks[c].num_dict_pages;
    chunks[c].page_info     = pages.device_ptr(page_count);
    page_count += chunks[c].max_num_pages;
  }

  chunks.host_to_device(stream);
  CUDA_TRY(gpu::DecodePageHeaders(chunks.device_ptr(), chunks.size(), stream));
  pages.device_to_host(stream, true);
}

/**
 * @copydoc cudf::io::detail::parquet::decompress_page_data
 */
rmm::device_buffer reader::impl::decompress_page_data(
  hostdevice_vector<gpu::ColumnChunkDesc> &chunks,
  hostdevice_vector<gpu::PageInfo> &pages,
  hipStream_t stream)
{
  auto for_each_codec_page = [&](parquet::Compression codec, const std::function<void(size_t)> &f) {
    for (size_t c = 0, page_count = 0; c < chunks.size(); c++) {
      const auto page_stride = chunks[c].max_num_pages;
      if (chunks[c].codec == codec) {
        for (int k = 0; k < page_stride; k++) { f(page_count + k); }
      }
      page_count += page_stride;
    }
  };

  // Brotli scratch memory for decompressing
  rmm::device_vector<uint8_t> debrotli_scratch;

  // Count the exact number of compressed pages
  size_t num_comp_pages    = 0;
  size_t total_decomp_size = 0;
  std::array<std::pair<parquet::Compression, size_t>, 3> codecs{std::make_pair(parquet::GZIP, 0),
                                                                std::make_pair(parquet::SNAPPY, 0),
                                                                std::make_pair(parquet::BROTLI, 0)};

  for (auto &codec : codecs) {
    for_each_codec_page(codec.first, [&](size_t page) {
      total_decomp_size += pages[page].uncompressed_page_size;
      codec.second++;
      num_comp_pages++;
    });
    if (codec.first == parquet::BROTLI && codec.second > 0) {
      debrotli_scratch.resize(get_gpu_debrotli_scratch_size(codec.second));
    }
  }

  // Dispatch batches of pages to decompress for each codec
  rmm::device_buffer decomp_pages(total_decomp_size, stream);
  hostdevice_vector<gpu_inflate_input_s> inflate_in(0, num_comp_pages, stream);
  hostdevice_vector<gpu_inflate_status_s> inflate_out(0, num_comp_pages, stream);

  size_t decomp_offset = 0;
  int32_t argc         = 0;
  for (const auto &codec : codecs) {
    if (codec.second > 0) {
      int32_t start_pos = argc;

      for_each_codec_page(codec.first, [&](size_t page) {
        auto dst_base              = static_cast<uint8_t *>(decomp_pages.data());
        inflate_in[argc].srcDevice = pages[page].page_data;
        inflate_in[argc].srcSize   = pages[page].compressed_page_size;
        inflate_in[argc].dstDevice = dst_base + decomp_offset;
        inflate_in[argc].dstSize   = pages[page].uncompressed_page_size;

        inflate_out[argc].bytes_written = 0;
        inflate_out[argc].status        = static_cast<uint32_t>(-1000);
        inflate_out[argc].reserved      = 0;

        pages[page].page_data = static_cast<uint8_t *>(inflate_in[argc].dstDevice);
        decomp_offset += inflate_in[argc].dstSize;
        argc++;
      });

      CUDA_TRY(hipMemcpyAsync(inflate_in.device_ptr(start_pos),
                               inflate_in.host_ptr(start_pos),
                               sizeof(decltype(inflate_in)::value_type) * (argc - start_pos),
                               hipMemcpyHostToDevice,
                               stream));
      CUDA_TRY(hipMemcpyAsync(inflate_out.device_ptr(start_pos),
                               inflate_out.host_ptr(start_pos),
                               sizeof(decltype(inflate_out)::value_type) * (argc - start_pos),
                               hipMemcpyHostToDevice,
                               stream));
      switch (codec.first) {
        case parquet::GZIP:
          CUDA_TRY(gpuinflate(inflate_in.device_ptr(start_pos),
                              inflate_out.device_ptr(start_pos),
                              argc - start_pos,
                              1,
                              stream))
          break;
        case parquet::SNAPPY:
          CUDA_TRY(gpu_unsnap(inflate_in.device_ptr(start_pos),
                              inflate_out.device_ptr(start_pos),
                              argc - start_pos,
                              stream));
          break;
        case parquet::BROTLI:
          CUDA_TRY(gpu_debrotli(inflate_in.device_ptr(start_pos),
                                inflate_out.device_ptr(start_pos),
                                debrotli_scratch.data().get(),
                                debrotli_scratch.size(),
                                argc - start_pos,
                                stream));
          break;
        default: CUDF_EXPECTS(false, "Unexpected decompression dispatch"); break;
      }
      CUDA_TRY(hipMemcpyAsync(inflate_out.host_ptr(start_pos),
                               inflate_out.device_ptr(start_pos),
                               sizeof(decltype(inflate_out)::value_type) * (argc - start_pos),
                               hipMemcpyDeviceToHost,
                               stream));
    }
  }
  CUDA_TRY(hipStreamSynchronize(stream));

  // Update the page information in device memory with the updated value of
  // page_data; it now points to the uncompressed data buffer
  CUDA_TRY(hipMemcpyAsync(
    pages.device_ptr(), pages.host_ptr(), pages.memory_size(), hipMemcpyHostToDevice, stream));

  return decomp_pages;
}

/**
 * @copydoc cudf::io::detail::parquet::allocate_nesting_info
 */
void reader::impl::allocate_nesting_info(hostdevice_vector<gpu::ColumnChunkDesc> const &chunks,
                                         hostdevice_vector<gpu::PageInfo> &pages,
                                         hostdevice_vector<gpu::PageNestingInfo> &page_nesting_info,
                                         hipStream_t stream)
{
  // compute total # of page_nesting infos needed and allocate space. doing this in one
  // buffer to keep it to a single gpu allocation
  size_t const total_page_nesting_infos = std::accumulate(
    chunks.host_ptr(), chunks.host_ptr() + chunks.size(), 0, [&](int total, auto &chunk) {
      // the schema of the input column
      auto const &schema                    = _metadata->get_schema(chunk.src_col_schema);
      auto const per_page_nesting_info_size = max(
        schema.max_definition_level + 1, _metadata->get_output_nesting_depth(chunk.src_col_schema));
      return total + (per_page_nesting_info_size * chunk.num_data_pages);
    });

  page_nesting_info = hostdevice_vector<gpu::PageNestingInfo>{total_page_nesting_infos, stream};

  // retrieve from the gpu so we can update
  pages.device_to_host(stream, true);

  // update pointers in the PageInfos
  int target_page_index = 0;
  int src_info_index    = 0;
  for (size_t idx = 0; idx < chunks.size(); idx++) {
    int src_col_schema = chunks[idx].src_col_schema;
    auto &schema       = _metadata->get_schema(src_col_schema);
    auto const per_page_nesting_info_size =
      max(schema.max_definition_level + 1, _metadata->get_output_nesting_depth(src_col_schema));

    // skip my dict pages
    target_page_index += chunks[idx].num_dict_pages;
    for (int p_idx = 0; p_idx < chunks[idx].num_data_pages; p_idx++) {
      pages[target_page_index + p_idx].nesting = page_nesting_info.device_ptr() + src_info_index;
      pages[target_page_index + p_idx].num_nesting_levels = per_page_nesting_info_size;

      src_info_index += per_page_nesting_info_size;
    }
    target_page_index += chunks[idx].num_data_pages;
  }

  // copy back to the gpu
  pages.host_to_device(stream);

  // fill in
  int nesting_info_index = 0;
  std::map<int, std::pair<std::vector<int>, std::vector<int>>> depth_remapping;
  for (size_t idx = 0; idx < chunks.size(); idx++) {
    int src_col_schema = chunks[idx].src_col_schema;

    // schema of the input column
    auto &schema = _metadata->get_schema(src_col_schema);
    // real depth of the output cudf column hierarchy (1 == no nesting, 2 == 1 level, etc)
    int max_depth = _metadata->get_output_nesting_depth(src_col_schema);

    // # of nesting infos stored per page for this column
    auto const per_page_nesting_info_size = max(schema.max_definition_level + 1, max_depth);

    // if this column has lists, generate depth remapping
    std::map<int, std::pair<std::vector<int>, std::vector<int>>> depth_remapping;
    if (schema.max_repetition_level > 0) {
      generate_depth_remappings(depth_remapping, src_col_schema, *_metadata);
    }

    // fill in host-side nesting info
    int schema_idx  = src_col_schema;
    auto cur_schema = _metadata->get_schema(schema_idx);
    int cur_depth   = max_depth - 1;
    while (schema_idx > 0) {
      // stub columns (basically the inner field of a list scheme element) are not real columns.
      // we can ignore them for the purposes of output nesting info
      if (!cur_schema.is_stub()) {
        // initialize each page within the chunk
        for (int p_idx = 0; p_idx < chunks[idx].num_data_pages; p_idx++) {
          gpu::PageNestingInfo *pni =
            &page_nesting_info[nesting_info_index + (p_idx * per_page_nesting_info_size)];

          // if we have lists, set our start and end depth remappings
          if (schema.max_repetition_level > 0) {
            auto remap = depth_remapping.find(src_col_schema);
            CUDF_EXPECTS(remap != depth_remapping.end(),
                         "Could not find depth remapping for schema");
            std::vector<int> const &rep_depth_remap = (remap->second.first);
            std::vector<int> const &def_depth_remap = (remap->second.second);

            for (size_t m = 0; m < rep_depth_remap.size(); m++) {
              pni[m].start_depth = rep_depth_remap[m];
            }
            for (size_t m = 0; m < def_depth_remap.size(); m++) {
              pni[m].end_depth = def_depth_remap[m];
            }
          }

          // values indexed by output column index
          pni[cur_depth].max_def_level = cur_schema.max_definition_level;
          pni[cur_depth].max_rep_level = cur_schema.max_repetition_level;
          pni[cur_depth].size          = 0;
        }

        // move up the hierarchy
        cur_depth--;
      }

      // next schema
      schema_idx = cur_schema.parent_idx;
      cur_schema = _metadata->get_schema(schema_idx);
    }

    nesting_info_index += (per_page_nesting_info_size * chunks[idx].num_data_pages);
  }

  // copy nesting info to the device
  page_nesting_info.host_to_device(stream);
}

/**
 * @copydoc cudf::io::detail::parquet::preprocess_columns
 */
void reader::impl::preprocess_columns(hostdevice_vector<gpu::ColumnChunkDesc> &chunks,
                                      hostdevice_vector<gpu::PageInfo> &pages,
                                      size_t min_row,
                                      size_t total_rows,
                                      bool has_lists,
                                      hipStream_t stream)
{
  // TODO : we should be selectively preprocessing only columns that have
  // lists in them instead of doing them all if even one contains lists.

  // if there are no lists, simply allocate every allocate every output
  // column to be of size num_rows
  if (!has_lists) {
    std::function<void(std::vector<column_buffer> &)> create_columns =
      [&](std::vector<column_buffer> &cols) {
        for (size_t idx = 0; idx < cols.size(); idx++) {
          auto &col = cols[idx];
          col.create(total_rows, stream, _mr);
          create_columns(col.children);
        }
      };
    create_columns(_output_columns);
  } else {
    // preprocess per-nesting level sizes by page
    CUDA_TRY(gpu::PreprocessColumnData(
      pages, chunks, _input_columns, _output_columns, total_rows, min_row, stream, _mr));
    CUDA_TRY(hipStreamSynchronize(stream));
  }
}

/**
 * @copydoc cudf::io::detail::parquet::decode_page_data
 */
void reader::impl::decode_page_data(hostdevice_vector<gpu::ColumnChunkDesc> &chunks,
                                    hostdevice_vector<gpu::PageInfo> &pages,
                                    hostdevice_vector<gpu::PageNestingInfo> &page_nesting,
                                    size_t min_row,
                                    size_t total_rows,
                                    hipStream_t stream)
{
  auto is_dict_chunk = [](const gpu::ColumnChunkDesc &chunk) {
    return (chunk.data_type & 0x7) == BYTE_ARRAY && chunk.num_dict_pages > 0;
  };

  // Count the number of string dictionary entries
  // NOTE: Assumes first page in the chunk is always the dictionary page
  size_t total_str_dict_indexes = 0;
  for (size_t c = 0, page_count = 0; c < chunks.size(); c++) {
    if (is_dict_chunk(chunks[c])) { total_str_dict_indexes += pages[page_count].num_input_values; }
    page_count += chunks[c].max_num_pages;
  }

  // Build index for string dictionaries since they can't be indexed
  // directly due to variable-sized elements
  rmm::device_vector<gpu::nvstrdesc_s> str_dict_index;
  if (total_str_dict_indexes > 0) { str_dict_index.resize(total_str_dict_indexes); }

  std::vector<hostdevice_vector<uint32_t *>> chunk_nested_valids;
  std::vector<hostdevice_vector<void *>> chunk_nested_data;

  // Update chunks with pointers to column data.
  for (size_t c = 0, page_count = 0, str_ofs = 0; c < chunks.size(); c++) {
    input_column_info const &input_col = _input_columns[chunks[c].src_col_index];
    CUDF_EXPECTS(input_col.schema_idx == chunks[c].src_col_schema,
                 "Column/page schema index mismatch");

    if (is_dict_chunk(chunks[c])) {
      chunks[c].str_dict_index = str_dict_index.data().get() + str_ofs;
      str_ofs += pages[page_count].num_input_values;
    }

    size_t max_depth = _metadata->get_output_nesting_depth(chunks[c].src_col_schema);

    // allocate (gpu) an array of pointers to validity data of size : nesting depth
    chunk_nested_valids.emplace_back(hostdevice_vector<uint32_t *>{max_depth});
    hostdevice_vector<uint32_t *> &valids = chunk_nested_valids.back();
    chunks[c].valid_map_base              = valids.device_ptr();

    // allocate (gpu) an array of pointers to out data of size : nesting depth
    chunk_nested_data.emplace_back(hostdevice_vector<void *>{max_depth});
    hostdevice_vector<void *> &data = chunk_nested_data.back();
    chunks[c].column_data_base      = data.device_ptr();

    // fill in the arrays on the host.  there are some important considerations to
    // take into account here for nested columns.  specifically, with structs
    // there is sharing of output buffers between input columns.  consider this schema
    //
    //  required group field_id=1 name {
    //    required binary field_id=2 firstname (String);
    //    required binary field_id=3 middlename (String);
    //    required binary field_id=4 lastname (String);
    // }
    //
    // there are 3 input columns of data here (firstname, middlename, lastname), but
    // only 1 output column (name).  The structure of the output column buffers looks like
    // the schema itself
    //
    // struct      (name)
    //     string  (firstname)
    //     string  (middlename)
    //     string  (lastname)
    //
    // The struct column can contain validity information. the problem is, the decode
    // step for the input columns will all attempt to decode this validity information
    // because each one has it's own copy of the repetition/definition levels. but
    // since this is all happening in parallel it would mean multiple blocks would
    // be stomping all over the same memory randomly.  to work around this, we set
    // things up so that only 1 child of any given nesting level fills in the
    // data (offsets in the case of lists) or validity information for the higher
    // levels of the hierarchy that are shared.  In this case, it would mean we
    // would just choose firstname to be the one that decodes the validity for name.
    //
    // we do this by only handing out the pointers to the first child we come across.
    //
    auto *cols = &_output_columns;
    for (size_t idx = 0; idx < max_depth; idx++) {
      auto &out_buf = (*cols)[input_col.nesting[idx]];
      cols          = &out_buf.children;

      int owning_schema = out_buf.user_data & PARQUET_COLUMN_BUFFER_SCHEMA_MASK;
      if (owning_schema == 0 || owning_schema == input_col.schema_idx) {
        valids[idx] = out_buf.null_mask();
        data[idx]   = out_buf.data();
        out_buf.user_data |=
          static_cast<uint32_t>(input_col.schema_idx) & PARQUET_COLUMN_BUFFER_SCHEMA_MASK;
      } else {
        valids[idx] = nullptr;
        data[idx]   = nullptr;
      }
    }

    // copy to the gpu
    valids.host_to_device(stream);
    data.host_to_device(stream);

    // column_data_base will always point to leaf data, even for nested types.
    page_count += chunks[c].max_num_pages;
  }

  chunks.host_to_device(stream);

  if (total_str_dict_indexes > 0) {
    CUDA_TRY(gpu::BuildStringDictionaryIndex(chunks.device_ptr(), chunks.size(), stream));
  }

  CUDA_TRY(gpu::DecodePageData(pages, chunks, total_rows, min_row, stream));
  pages.device_to_host(stream);
  page_nesting.device_to_host(stream);
  hipStreamSynchronize(stream);

  // for list columns, add the final offset to every offset buffer.
  // TODO : make this happen in more efficiently. Maybe use thrust::for_each
  // on each buffer.  Or potentially do it in PreprocessColumnData
  // Note : the reason we are doing this here instead of in the decode kernel is
  // that it is difficult/impossible for a given page to know that it is writing the very
  // last value that should then be followed by a terminator (because rows can span
  // page boundaries).
  for (size_t idx = 0; idx < _input_columns.size(); idx++) {
    input_column_info const &input_col = _input_columns[idx];

    auto *cols = &_output_columns;
    for (size_t l_idx = 0; l_idx < input_col.nesting_depth(); l_idx++) {
      auto &out_buf = (*cols)[input_col.nesting[l_idx]];
      cols          = &out_buf.children;

      if (out_buf.type.id() != type_id::LIST ||
          (out_buf.user_data & PARQUET_COLUMN_BUFFER_FLAG_LIST_TERMINATED)) {
        continue;
      }
      CUDF_EXPECTS(l_idx < input_col.nesting_depth() - 1, "Encountered a leaf list column");
      auto &child = (*cols)[input_col.nesting[l_idx + 1]];

      // the final offset for a list at level N is the size of it's child
      int offset = child.type.id() == type_id::LIST ? child.size - 1 : child.size;
      hipMemcpyAsync(static_cast<int32_t *>(out_buf.data()) + (out_buf.size - 1),
                      &offset,
                      sizeof(offset),
                      hipMemcpyHostToDevice,
                      stream);
      out_buf.user_data |= PARQUET_COLUMN_BUFFER_FLAG_LIST_TERMINATED;
    }
  }

  // update null counts in the final column buffers
  for (size_t idx = 0; idx < pages.size(); idx++) {
    gpu::PageInfo *pi = &pages[idx];
    if (pi->flags & gpu::PAGEINFO_FLAGS_DICTIONARY) { continue; }
    gpu::ColumnChunkDesc *col          = &chunks[pi->chunk_idx];
    input_column_info const &input_col = _input_columns[col->src_col_index];

    int index                 = pi->nesting - page_nesting.device_ptr();
    gpu::PageNestingInfo *pni = &page_nesting[index];

    auto *cols = &_output_columns;
    for (size_t l_idx = 0; l_idx < input_col.nesting_depth(); l_idx++) {
      auto &out_buf = (*cols)[input_col.nesting[l_idx]];
      cols          = &out_buf.children;

      // if I wasn't the one who wrote out the validity bits, skip it
      if (chunk_nested_valids[pi->chunk_idx][l_idx] == nullptr) { continue; }
      out_buf.null_count() += pni[l_idx].value_count - pni[l_idx].valid_count;
    }
  }

  hipStreamSynchronize(stream);
}

reader::impl::impl(std::vector<std::unique_ptr<datasource>> &&sources,
                   parquet_reader_options const &options,
                   rmm::mr::device_memory_resource *mr)
  : _sources(std::move(sources)), _mr(mr)
{
  // Open and parse the source dataset metadata
  _metadata = std::make_unique<aggregate_metadata>(_sources);

  // Override output timestamp resolution if requested
  if (options.get_timestamp_type().id() != type_id::EMPTY) {
    _timestamp_type = options.get_timestamp_type();
  }

  // Strings may be returned as either string or categorical columns
  _strings_to_categorical = options.is_enabled_convert_strings_to_categories();

  // Select only columns required by the options
  std::tie(_input_columns, _output_columns, _output_column_schemas) =
    _metadata->select_columns(options.get_columns(),
                              options.is_enabled_use_pandas_metadata(),
                              _strings_to_categorical,
                              _timestamp_type.id());
}

table_with_metadata reader::impl::read(size_type skip_rows,
                                       size_type num_rows,
                                       std::vector<std::vector<size_type>> const &row_group_list,
                                       hipStream_t stream)
{
  // Select only row groups required
  const auto selected_row_groups =
    _metadata->select_row_groups(row_group_list, skip_rows, num_rows);

  table_metadata out_metadata;

  // output cudf columns as determined by the top level schema
  std::vector<std::unique_ptr<column>> out_columns;
  out_columns.reserve(_output_columns.size());

  if (selected_row_groups.size() != 0 && _input_columns.size() != 0) {
    // Descriptors for all the chunks that make up the selected columns
    const auto num_input_columns = _input_columns.size();
    const auto num_chunks        = selected_row_groups.size() * num_input_columns;
    hostdevice_vector<gpu::ColumnChunkDesc> chunks(0, num_chunks, stream);

    // Association between each column chunk and its source
    std::vector<size_type> chunk_source_map(num_chunks);

    // Tracker for eventually deallocating compressed and uncompressed data
    std::vector<rmm::device_buffer> page_data(num_chunks);

    // Keep track of column chunk file offsets
    std::vector<size_t> column_chunk_offsets(num_chunks);

    // if there are lists present, we need to preprocess
    bool has_lists = false;

    // Initialize column chunk information
    size_t total_decompressed_size = 0;
    auto remaining_rows            = num_rows;
    for (const auto &rg : selected_row_groups) {
      const auto &row_group       = _metadata->get_row_group(rg.index, rg.source_index);
      auto const row_group_start  = rg.start_row;
      auto const row_group_source = rg.source_index;
      auto const row_group_rows   = std::min<int>(remaining_rows, row_group.num_rows);
      auto const io_chunk_idx     = chunks.size();

      // generate ColumnChunkDesc objects for everything to be decoded (all input columns)
      for (size_t i = 0; i < num_input_columns; ++i) {
        auto col = _input_columns[i];
        // look up metadata
        auto &col_meta = _metadata->get_column_metadata(rg.index, rg.source_index, col.schema_idx);
        auto &schema   = _metadata->get_schema(col.schema_idx);

        // this column contains repetition levels and will require a preprocess
        if (schema.max_repetition_level > 0) { has_lists = true; }

        // Spec requires each row group to contain exactly one chunk for every
        // column. If there are too many or too few, continue with best effort
        if (chunks.size() >= chunks.max_size()) {
          std::cerr << "Detected too many column chunks" << std::endl;
          continue;
        }

        int32_t type_width;
        int32_t clock_rate;
        int8_t converted_type;

        std::tie(type_width, clock_rate, converted_type) =
          conversion_info(to_type_id(schema, _strings_to_categorical, _timestamp_type.id()),
                          _timestamp_type.id(),
                          schema.type,
                          schema.converted_type,
                          schema.type_length);

        column_chunk_offsets[chunks.size()] =
          (col_meta.dictionary_page_offset != 0)
            ? std::min(col_meta.data_page_offset, col_meta.dictionary_page_offset)
            : col_meta.data_page_offset;

        chunks.insert(gpu::ColumnChunkDesc(col_meta.total_compressed_size,
                                           nullptr,
                                           col_meta.num_values,
                                           schema.type,
                                           type_width,
                                           row_group_start,
                                           row_group_rows,
                                           schema.max_definition_level,
                                           schema.max_repetition_level,
                                           _metadata->get_output_nesting_depth(col.schema_idx),
                                           required_bits(schema.max_definition_level),
                                           required_bits(schema.max_repetition_level),
                                           col_meta.codec,
                                           converted_type,
                                           schema.decimal_scale,
                                           clock_rate,
                                           i,
                                           col.schema_idx));

        // Map each column chunk to its column index and its source index
        chunk_source_map[chunks.size() - 1] = row_group_source;

        if (col_meta.codec != Compression::UNCOMPRESSED) {
          total_decompressed_size += col_meta.total_uncompressed_size;
        }
      }
      // Read compressed chunk data to device memory
      read_column_chunks(page_data,
                         chunks,
                         io_chunk_idx,
                         chunks.size(),
                         column_chunk_offsets,
                         chunk_source_map,
                         stream);

      remaining_rows -= row_group.num_rows;
    }
    assert(remaining_rows <= 0);

    // Process dataset chunk pages into output columns
    const auto total_pages = count_page_headers(chunks, stream);
    if (total_pages > 0) {
      hostdevice_vector<gpu::PageInfo> pages(total_pages, total_pages, stream);
      rmm::device_buffer decomp_page_data;

      // decoding of column/page information
      decode_page_headers(chunks, pages, stream);
      if (total_decompressed_size > 0) {
        decomp_page_data = decompress_page_data(chunks, pages, stream);
        // Free compressed data
        for (size_t c = 0; c < chunks.size(); c++) {
          if (chunks[c].codec != parquet::Compression::UNCOMPRESSED && page_data[c].size() != 0) {
            page_data[c].resize(0);
            page_data[c].shrink_to_fit();
          }
        }
      }

      // build output column info
      // walk the schema, building out_buffers that mirror what our final cudf columns will look
      // like. important : there is not necessarily a 1:1 mapping between input columns and output
      // columns. For example, parquet does not explicitly store a ColumnChunkDesc for struct
      // columns. The "structiness" is simply implied by the schema.  For example, this schema:
      //  required group field_id=1 name {
      //    required binary field_id=2 firstname (String);
      //    required binary field_id=3 middlename (String);
      //    required binary field_id=4 lastname (String);
      // }
      // will only contain 3 columns of data (firstname, middlename, lastname).  But of course
      // "name" is a struct column that we want to return, so we have to make sure that we
      // create it ourselves.
      // std::vector<output_column_info> output_info = build_output_column_info();

      // nesting information (sizes, etc) stored -per page-
      // note : even for flat schemas, we allocate 1 level of "nesting" info
      hostdevice_vector<gpu::PageNestingInfo> page_nesting_info;
      allocate_nesting_info(chunks, pages, page_nesting_info, stream);

      // - compute column sizes and allocate output buffers.
      //   important:
      //   for nested schemas, we have to do some further preprocessing to determine:
      //    - real column output sizes per level of nesting (in a flat schema, there's only 1 level
      //    of
      //      nesting and it's size is the row count)
      //
      // - for nested schemas, output buffer offset values per-page, per nesting-level for the
      // purposes of decoding.
      preprocess_columns(chunks, pages, skip_rows, num_rows, has_lists, stream);

      // decoding of column data itself
      decode_page_data(chunks, pages, page_nesting_info, skip_rows, num_rows, stream);

      // create the final output cudf columns
      for (size_t i = 0; i < _output_columns.size(); ++i) {
        out_metadata.schema_info.push_back(column_name_info{""});
        out_columns.emplace_back(
          make_column(_output_columns[i], stream, _mr, &out_metadata.schema_info.back()));
      }
    }
  }

  // Create empty columns as needed (this can happen if we've ended up with no actual data to read)
  for (size_t i = out_columns.size(); i < _output_columns.size(); ++i) {
    out_metadata.schema_info.push_back(column_name_info{""});
    out_columns.emplace_back(make_empty_column(_output_columns[i].type));
  }

  // Return column names (must match order of returned columns)
  out_metadata.column_names.resize(_output_columns.size());
  for (size_t i = 0; i < _output_column_schemas.size(); i++) {
    auto const &schema           = _metadata->get_schema(_output_column_schemas[i]);
    out_metadata.column_names[i] = schema.name;
  }

  // Return user metadata
  out_metadata.user_data = _metadata->get_key_value_metadata();

  return {std::make_unique<table>(std::move(out_columns)), std::move(out_metadata)};
}

// Forward to implementation
reader::reader(std::vector<std::string> const &filepaths,
               parquet_reader_options const &options,
               rmm::mr::device_memory_resource *mr)
  : _impl(std::make_unique<impl>(datasource::create(filepaths), options, mr))
{
}

// Forward to implementation
reader::reader(std::vector<std::unique_ptr<cudf::io::datasource>> &&sources,
               parquet_reader_options const &options,
               rmm::mr::device_memory_resource *mr)
  : _impl(std::make_unique<impl>(std::move(sources), options, mr))
{
}

// Destructor within this translation unit
reader::~reader() = default;

// Forward to implementation
table_with_metadata reader::read(parquet_reader_options const &options, hipStream_t stream)
{
  return _impl->read(
    options.get_skip_rows(), options.get_num_rows(), options.get_row_groups(), stream);
}

}  // namespace parquet
}  // namespace detail
}  // namespace io
}  // namespace cudf
