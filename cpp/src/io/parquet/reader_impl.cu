#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019-2022, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

/**
 * @file reader_impl.cu
 * @brief cuDF-IO Parquet reader class implementation
 */

#include "reader_impl.hpp"

#include "compact_protocol_reader.hpp"

#include <io/comp/gpuinflate.hpp>
#include <io/comp/nvcomp_adapter.hpp>
#include <io/utilities/config_utils.hpp>
#include <io/utilities/time_utils.cuh>

#include <cudf/detail/utilities/integer_utils.hpp>
#include <cudf/detail/utilities/vector_factories.hpp>
#include <cudf/table/table.hpp>
#include <cudf/utilities/error.hpp>
#include <cudf/utilities/traits.hpp>

#include <rmm/cuda_stream_view.hpp>
#include <rmm/device_buffer.hpp>
#include <rmm/device_uvector.hpp>
#include <rmm/exec_policy.hpp>

#include <thrust/fill.h>
#include <thrust/for_each.h>
#include <thrust/iterator/zip_iterator.h>
#include <thrust/logical.h>
#include <thrust/transform.h>
#include <thrust/tuple.h>

#include <algorithm>
#include <array>
#include <numeric>
#include <regex>

namespace cudf {
namespace io {
namespace detail {
namespace parquet {
// Import functionality that's independent of legacy code
using namespace cudf::io::parquet;
using namespace cudf::io;

namespace {

parquet::ConvertedType logical_type_to_converted_type(parquet::LogicalType const& logical)
{
  if (logical.isset.STRING) {
    return parquet::UTF8;
  } else if (logical.isset.MAP) {
    return parquet::MAP;
  } else if (logical.isset.LIST) {
    return parquet::LIST;
  } else if (logical.isset.ENUM) {
    return parquet::ENUM;
  } else if (logical.isset.DECIMAL) {
    return parquet::DECIMAL;  // TODO set decimal values
  } else if (logical.isset.DATE) {
    return parquet::DATE;
  } else if (logical.isset.TIME) {
    if (logical.TIME.unit.isset.MILLIS)
      return parquet::TIME_MILLIS;
    else if (logical.TIME.unit.isset.MICROS)
      return parquet::TIME_MICROS;
  } else if (logical.isset.TIMESTAMP) {
    if (logical.TIMESTAMP.unit.isset.MILLIS)
      return parquet::TIMESTAMP_MILLIS;
    else if (logical.TIMESTAMP.unit.isset.MICROS)
      return parquet::TIMESTAMP_MICROS;
  } else if (logical.isset.INTEGER) {
    switch (logical.INTEGER.bitWidth) {
      case 8: return logical.INTEGER.isSigned ? INT_8 : UINT_8;
      case 16: return logical.INTEGER.isSigned ? INT_16 : UINT_16;
      case 32: return logical.INTEGER.isSigned ? INT_32 : UINT_32;
      case 64: return logical.INTEGER.isSigned ? INT_64 : UINT_64;
      default: break;
    }
  } else if (logical.isset.UNKNOWN) {
    return parquet::NA;
  } else if (logical.isset.JSON) {
    return parquet::JSON;
  } else if (logical.isset.BSON) {
    return parquet::BSON;
  }
  return parquet::UNKNOWN;
}

/**
 * @brief Function that translates Parquet datatype to cuDF type enum
 */
type_id to_type_id(SchemaElement const& schema,
                   bool strings_to_categorical,
                   type_id timestamp_type_id)
{
  parquet::Type const physical            = schema.type;
  parquet::LogicalType const logical_type = schema.logical_type;
  parquet::ConvertedType converted_type   = schema.converted_type;
  int32_t decimal_scale                   = schema.decimal_scale;

  // Logical type used for actual data interpretation; the legacy converted type
  // is superceded by 'logical' type whenever available.
  auto const inferred_converted_type = logical_type_to_converted_type(logical_type);
  if (inferred_converted_type != parquet::UNKNOWN) converted_type = inferred_converted_type;
  if (inferred_converted_type == parquet::DECIMAL && decimal_scale == 0)
    decimal_scale = schema.logical_type.DECIMAL.scale;

  switch (converted_type) {
    case parquet::UINT_8: return type_id::UINT8;
    case parquet::INT_8: return type_id::INT8;
    case parquet::UINT_16: return type_id::UINT16;
    case parquet::INT_16: return type_id::INT16;
    case parquet::UINT_32: return type_id::UINT32;
    case parquet::UINT_64: return type_id::UINT64;
    case parquet::DATE: return type_id::TIMESTAMP_DAYS;
    case parquet::TIME_MILLIS:
      return (timestamp_type_id != type_id::EMPTY) ? timestamp_type_id
                                                   : type_id::DURATION_MILLISECONDS;
    case parquet::TIME_MICROS:
      return (timestamp_type_id != type_id::EMPTY) ? timestamp_type_id
                                                   : type_id::DURATION_MICROSECONDS;
    case parquet::TIMESTAMP_MILLIS:
      return (timestamp_type_id != type_id::EMPTY) ? timestamp_type_id
                                                   : type_id::TIMESTAMP_MILLISECONDS;
    case parquet::TIMESTAMP_MICROS:
      return (timestamp_type_id != type_id::EMPTY) ? timestamp_type_id
                                                   : type_id::TIMESTAMP_MICROSECONDS;
    case parquet::DECIMAL:
      if (physical == parquet::INT32) { return type_id::DECIMAL32; }
      if (physical == parquet::INT64) { return type_id::DECIMAL64; }
      if (physical == parquet::FIXED_LEN_BYTE_ARRAY) {
        if (schema.type_length <= static_cast<int32_t>(sizeof(int32_t))) {
          return type_id::DECIMAL32;
        }
        if (schema.type_length <= static_cast<int32_t>(sizeof(int64_t))) {
          return type_id::DECIMAL64;
        }
        if (schema.type_length <= static_cast<int32_t>(sizeof(__int128_t))) {
          return type_id::DECIMAL128;
        }
      }
      CUDF_FAIL("Invalid representation of decimal type");
      break;

    // maps are just List<Struct<>>.
    case parquet::MAP:
    case parquet::LIST: return type_id::LIST;
    case parquet::NA: return type_id::STRING;
    // return type_id::EMPTY; //TODO(kn): enable after Null/Empty column support
    default: break;
  }

  if (inferred_converted_type == parquet::UNKNOWN and physical == parquet::INT64 and
      logical_type.TIMESTAMP.unit.isset.NANOS) {
    return (timestamp_type_id != type_id::EMPTY) ? timestamp_type_id
                                                 : type_id::TIMESTAMP_NANOSECONDS;
  }

  // is it simply a struct?
  if (schema.is_struct()) { return type_id::STRUCT; }

  // Physical storage type supported by Parquet; controls the on-disk storage
  // format in combination with the encoding type.
  switch (physical) {
    case parquet::BOOLEAN: return type_id::BOOL8;
    case parquet::INT32: return type_id::INT32;
    case parquet::INT64: return type_id::INT64;
    case parquet::FLOAT: return type_id::FLOAT32;
    case parquet::DOUBLE: return type_id::FLOAT64;
    case parquet::BYTE_ARRAY:
    case parquet::FIXED_LEN_BYTE_ARRAY:
      // Can be mapped to INT32 (32-bit hash) or STRING
      return strings_to_categorical ? type_id::INT32 : type_id::STRING;
    case parquet::INT96:
      return (timestamp_type_id != type_id::EMPTY) ? timestamp_type_id
                                                   : type_id::TIMESTAMP_NANOSECONDS;
    default: break;
  }

  return type_id::EMPTY;
}

/**
 * @brief Converts cuDF type enum to column logical type
 */
data_type to_data_type(type_id t_id, SchemaElement const& schema)
{
  return t_id == type_id::DECIMAL32 || t_id == type_id::DECIMAL64 || t_id == type_id::DECIMAL128
           ? data_type{t_id, numeric::scale_type{-schema.decimal_scale}}
           : data_type{t_id};
}

/**
 * @brief Function that returns the required the number of bits to store a value
 */
template <typename T = uint8_t>
T required_bits(uint32_t max_level)
{
  return static_cast<T>(CompactProtocolReader::NumRequiredBits(max_level));
}

/**
 * @brief Converts cuDF units to Parquet units.
 *
 * @return A tuple of Parquet type width, Parquet clock rate and Parquet decimal type.
 */
std::tuple<int32_t, int32_t, int8_t> conversion_info(type_id column_type_id,
                                                     type_id timestamp_type_id,
                                                     parquet::Type physical,
                                                     int8_t converted,
                                                     int32_t length)
{
  int32_t type_width = (physical == parquet::FIXED_LEN_BYTE_ARRAY) ? length : 0;
  int32_t clock_rate = 0;
  if (column_type_id == type_id::INT8 or column_type_id == type_id::UINT8) {
    type_width = 1;  // I32 -> I8
  } else if (column_type_id == type_id::INT16 or column_type_id == type_id::UINT16) {
    type_width = 2;  // I32 -> I16
  } else if (column_type_id == type_id::INT32) {
    type_width = 4;  // str -> hash32
  } else if (is_chrono(data_type{column_type_id})) {
    clock_rate = to_clockrate(timestamp_type_id);
  }

  int8_t converted_type = converted;
  if (converted_type == parquet::DECIMAL && column_type_id != type_id::FLOAT64 &&
      not cudf::is_fixed_point(column_type_id)) {
    converted_type = parquet::UNKNOWN;  // Not converting to float64 or decimal
  }
  return std::make_tuple(type_width, clock_rate, converted_type);
}

inline void decompress_check(device_span<decompress_status const> stats,
                             rmm::cuda_stream_view stream)
{
  CUDF_EXPECTS(thrust::all_of(rmm::exec_policy(stream),
                              stats.begin(),
                              stats.end(),
                              [] __device__(auto const& stat) { return stat.status == 0; }),
               "Error during decompression");
}
}  // namespace

std::string name_from_path(const std::vector<std::string>& path_in_schema)
{
  // For the case of lists, we will see a schema that looks like:
  // a.list.element.list.element
  // where each (list.item) pair represents a level of nesting.  According to the parquet spec,
  // https://github.com/apache/parquet-format/blob/master/LogicalTypes.md
  // the initial field must be named "list" and the inner element must be named "element".
  // If we are dealing with a list, we want to return the topmost name of the group ("a").
  //
  // For other nested schemas, like structs we just want to return the bottom-most name. For
  // example a struct with the schema
  // b.employee.id,  the column representing "id" should simply be named "id".
  //
  // In short, this means : return the highest level of the schema that does not have list
  // definitions underneath it.
  //
  std::string s = (path_in_schema.size() > 0) ? path_in_schema[0] : "";
  for (size_t i = 1; i < path_in_schema.size(); i++) {
    // The Parquet spec requires that the outer schema field is named "list". However it also
    // provides a list of backwards compatibility cases that are applicable as well.  Currently
    // we are only handling the formal spec.  This will get cleaned up and improved when we add
    // support for structs. The correct thing to do will probably be to examine the type of
    // the SchemaElement itself to concretely identify the start of a nested type of any kind rather
    // than trying to derive it from the path string.
    if (path_in_schema[i] == "list") {
      // Again, strictly speaking, the Parquet spec says the inner field should be named
      // "element", but there are some backwards compatibility issues that we have seen in the
      // wild. For example, Pandas calls the field "item".  We will allow any name for now.
      i++;
      continue;
    }
    // otherwise, we've got a real nested column. update the name
    s = path_in_schema[i];
  }
  return s;
}

/**
 * @brief Class for parsing dataset metadata
 */
struct metadata : public FileMetaData {
  explicit metadata(datasource* source)
  {
    constexpr auto header_len = sizeof(file_header_s);
    constexpr auto ender_len  = sizeof(file_ender_s);

    const auto len           = source->size();
    const auto header_buffer = source->host_read(0, header_len);
    const auto header        = reinterpret_cast<const file_header_s*>(header_buffer->data());
    const auto ender_buffer  = source->host_read(len - ender_len, ender_len);
    const auto ender         = reinterpret_cast<const file_ender_s*>(ender_buffer->data());
    CUDF_EXPECTS(len > header_len + ender_len, "Incorrect data source");
    CUDF_EXPECTS(header->magic == parquet_magic && ender->magic == parquet_magic,
                 "Corrupted header or footer");
    CUDF_EXPECTS(ender->footer_len != 0 && ender->footer_len <= (len - header_len - ender_len),
                 "Incorrect footer length");

    const auto buffer = source->host_read(len - ender->footer_len - ender_len, ender->footer_len);
    CompactProtocolReader cp(buffer->data(), ender->footer_len);
    CUDF_EXPECTS(cp.read(this), "Cannot parse metadata");
    CUDF_EXPECTS(cp.InitSchema(this), "Cannot initialize schema");
  }
};

class aggregate_reader_metadata {
  std::vector<metadata> per_file_metadata;
  std::vector<std::unordered_map<std::string, std::string>> keyval_maps;
  size_type num_rows;
  size_type num_row_groups;
  /**
   * @brief Create a metadata object from each element in the source vector
   */
  auto metadatas_from_sources(std::vector<std::unique_ptr<datasource>> const& sources)
  {
    std::vector<metadata> metadatas;
    std::transform(
      sources.cbegin(), sources.cend(), std::back_inserter(metadatas), [](auto const& source) {
        return metadata(source.get());
      });
    return metadatas;
  }

  /**
   * @brief Collect the keyvalue maps from each per-file metadata object into a vector of maps.
   */
  [[nodiscard]] auto collect_keyval_metadata()
  {
    std::vector<std::unordered_map<std::string, std::string>> kv_maps;
    std::transform(per_file_metadata.cbegin(),
                   per_file_metadata.cend(),
                   std::back_inserter(kv_maps),
                   [](auto const& pfm) {
                     std::unordered_map<std::string, std::string> kv_map;
                     std::transform(pfm.key_value_metadata.cbegin(),
                                    pfm.key_value_metadata.cend(),
                                    std::inserter(kv_map, kv_map.end()),
                                    [](auto const& kv) {
                                      return std::pair{kv.key, kv.value};
                                    });
                     return kv_map;
                   });

    return kv_maps;
  }

  /**
   * @brief Sums up the number of rows of each source
   */
  [[nodiscard]] size_type calc_num_rows() const
  {
    return std::accumulate(
      per_file_metadata.begin(), per_file_metadata.end(), 0, [](auto& sum, auto& pfm) {
        return sum + pfm.num_rows;
      });
  }

  /**
   * @brief Sums up the number of row groups of each source
   */
  [[nodiscard]] size_type calc_num_row_groups() const
  {
    return std::accumulate(
      per_file_metadata.begin(), per_file_metadata.end(), 0, [](auto& sum, auto& pfm) {
        return sum + pfm.row_groups.size();
      });
  }

 public:
  aggregate_reader_metadata(std::vector<std::unique_ptr<datasource>> const& sources)
    : per_file_metadata(metadatas_from_sources(sources)),
      keyval_maps(collect_keyval_metadata()),
      num_rows(calc_num_rows()),
      num_row_groups(calc_num_row_groups())
  {
    // Verify that the input files have matching numbers of columns
    size_type num_cols = -1;
    for (auto const& pfm : per_file_metadata) {
      if (pfm.row_groups.size() != 0) {
        if (num_cols == -1)
          num_cols = pfm.row_groups[0].columns.size();
        else
          CUDF_EXPECTS(num_cols == static_cast<size_type>(pfm.row_groups[0].columns.size()),
                       "All sources must have the same number of columns");
      }
    }
    // Verify that the input files have matching schemas
    for (auto const& pfm : per_file_metadata) {
      CUDF_EXPECTS(per_file_metadata[0].schema == pfm.schema,
                   "All sources must have the same schemas");
    }
  }

  [[nodiscard]] auto const& get_row_group(size_type row_group_index, size_type src_idx) const
  {
    CUDF_EXPECTS(src_idx >= 0 && src_idx < static_cast<size_type>(per_file_metadata.size()),
                 "invalid source index");
    return per_file_metadata[src_idx].row_groups[row_group_index];
  }

  [[nodiscard]] auto const& get_column_metadata(size_type row_group_index,
                                                size_type src_idx,
                                                int schema_idx) const
  {
    auto col = std::find_if(
      per_file_metadata[src_idx].row_groups[row_group_index].columns.begin(),
      per_file_metadata[src_idx].row_groups[row_group_index].columns.end(),
      [schema_idx](ColumnChunk const& col) { return col.schema_idx == schema_idx ? true : false; });
    CUDF_EXPECTS(col != std::end(per_file_metadata[src_idx].row_groups[row_group_index].columns),
                 "Found no metadata for schema index");
    return col->meta_data;
  }

  [[nodiscard]] auto get_num_rows() const { return num_rows; }

  [[nodiscard]] auto get_num_row_groups() const { return num_row_groups; }

  [[nodiscard]] auto const& get_schema(int schema_idx) const
  {
    return per_file_metadata[0].schema[schema_idx];
  }

  [[nodiscard]] auto const& get_key_value_metadata() const { return keyval_maps; }

  /**
   * @brief Gets the concrete nesting depth of output cudf columns
   *
   * @param schema_index Schema index of the input column
   *
   * @return comma-separated index column names in quotes
   */
  [[nodiscard]] inline int get_output_nesting_depth(int schema_index) const
  {
    auto& pfm = per_file_metadata[0];
    int depth = 0;

    // walk upwards, skipping repeated fields
    while (schema_index > 0) {
      if (!pfm.schema[schema_index].is_stub()) { depth++; }
      // schema of one-level encoding list doesn't contain nesting information, so we need to
      // manually add an extra nesting level
      if (pfm.schema[schema_index].is_one_level_list()) { depth++; }
      schema_index = pfm.schema[schema_index].parent_idx;
    }
    return depth;
  }

  /**
   * @brief Extracts the pandas "index_columns" section
   *
   * PANDAS adds its own metadata to the key_value section when writing out the
   * dataframe to a file to aid in exact reconstruction. The JSON-formatted
   * metadata contains the index column(s) and PANDA-specific datatypes.
   *
   * @return comma-separated index column names in quotes
   */
  [[nodiscard]] std::string get_pandas_index() const
  {
    // Assumes that all input files have the same metadata
    // TODO: verify this assumption
    auto it = keyval_maps[0].find("pandas");
    if (it != keyval_maps[0].end()) {
      // Captures a list of quoted strings found inside square brackets after `"index_columns":`
      // Inside quotes supports newlines, brackets, escaped quotes, etc.
      // One-liner regex:
      // "index_columns"\s*:\s*\[\s*((?:"(?:|(?:.*?(?![^\\]")).?)[^\\]?",?\s*)*)\]
      // Documented below.
      std::regex index_columns_expr{
        R"("index_columns"\s*:\s*\[\s*)"  // match preamble, opening square bracket, whitespace
        R"(()"                            // Open first capturing group
        R"((?:")"                         // Open non-capturing group match opening quote
        R"((?:|(?:.*?(?![^\\]")).?))"     // match empty string or anything between quotes
        R"([^\\]?")"                      // Match closing non-escaped quote
        R"(,?\s*)"                        // Match optional comma and whitespace
        R"()*)"                           // Close non-capturing group and repeat 0 or more times
        R"())"                            // Close first capturing group
        R"(\])"                           // Match closing square brackets
      };
      std::smatch sm;
      if (std::regex_search(it->second, sm, index_columns_expr)) { return sm[1].str(); }
    }
    return "";
  }

  /**
   * @brief Extracts the column name(s) used for the row indexes in a dataframe
   *
   * @param names List of column names to load, where index column name(s) will be added
   */
  [[nodiscard]] std::vector<std::string> get_pandas_index_names() const
  {
    std::vector<std::string> names;
    auto str = get_pandas_index();
    if (str.length() != 0) {
      std::regex index_name_expr{R"(\"((?:\\.|[^\"])*)\")"};
      std::smatch sm;
      while (std::regex_search(str, sm, index_name_expr)) {
        if (sm.size() == 2) {  // 2 = whole match, first item
          if (std::find(names.begin(), names.end(), sm[1].str()) == names.end()) {
            std::regex esc_quote{R"(\\")"};
            names.emplace_back(std::regex_replace(sm[1].str(), esc_quote, R"(")"));
          }
        }
        str = sm.suffix();
      }
    }
    return names;
  }

  struct row_group_info {
    size_type const index;
    size_t const start_row;  // TODO source index
    size_type const source_index;
    row_group_info(size_type index, size_t start_row, size_type source_index)
      : index(index), start_row(start_row), source_index(source_index)
    {
    }
  };

  /**
   * @brief Filters and reduces down to a selection of row groups
   *
   * @param row_groups Lists of row groups to read, one per source
   *
   * @return List of row group info structs and the total number of rows
   */
  [[nodiscard]] std::pair<std::vector<row_group_info>, size_type> select_row_groups(
    std::vector<std::vector<size_type>> const& row_groups) const
  {
    size_type row_count = 0;

    if (!row_groups.empty()) {
      std::vector<row_group_info> selection;
      CUDF_EXPECTS(row_groups.size() == per_file_metadata.size(),
                   "Must specify row groups for each source");

      row_count = 0;
      for (size_t src_idx = 0; src_idx < row_groups.size(); ++src_idx) {
        for (auto const& rowgroup_idx : row_groups[src_idx]) {
          CUDF_EXPECTS(
            rowgroup_idx >= 0 &&
              rowgroup_idx < static_cast<size_type>(per_file_metadata[src_idx].row_groups.size()),
            "Invalid rowgroup index");
          selection.emplace_back(rowgroup_idx, row_count, src_idx);
          row_count += get_row_group(rowgroup_idx, src_idx).num_rows;
        }
      }
      return {selection, row_count};
    }

    row_count = static_cast<size_type>(
      std::min<int64_t>(get_num_rows(), std::numeric_limits<size_type>::max()));
    CUDF_EXPECTS(row_count >= 0, "Invalid row count");

    std::vector<row_group_info> selection;
    size_type count = 0;
    for (size_t src_idx = 0; src_idx < per_file_metadata.size(); ++src_idx) {
      for (size_t rg_idx = 0; rg_idx < per_file_metadata[src_idx].row_groups.size(); ++rg_idx) {
        auto const chunk_start_row = count;
        count += get_row_group(rg_idx, src_idx).num_rows;
        selection.emplace_back(rg_idx, chunk_start_row, src_idx);
        if (count >= row_count) { break; }
      }
    }

    return {selection, row_count};
  }

  /**
   * @brief Filters and reduces down to a selection of columns
   *
   * @param use_names List of paths of column names to select; `nullopt` if user did not select
   * columns to read
   * @param include_index Whether to always include the PANDAS index column(s)
   * @param strings_to_categorical Type conversion parameter
   * @param timestamp_type_id Type conversion parameter
   *
   * @return input column information, output column information, list of output column schema
   * indices
   */
  [[nodiscard]] auto select_columns(std::optional<std::vector<std::string>> const& use_names,
                                    bool include_index,
                                    bool strings_to_categorical,
                                    type_id timestamp_type_id) const
  {
    auto find_schema_child = [&](SchemaElement const& schema_elem, std::string const& name) {
      auto const& col_schema_idx = std::find_if(
        schema_elem.children_idx.cbegin(),
        schema_elem.children_idx.cend(),
        [&](size_t col_schema_idx) { return get_schema(col_schema_idx).name == name; });

      return (col_schema_idx != schema_elem.children_idx.end()) ? static_cast<int>(*col_schema_idx)
                                                                : -1;
    };

    std::vector<column_buffer> output_columns;
    std::vector<input_column_info> input_columns;
    std::vector<int> nesting;

    // Return true if column path is valid. e.g. if the path is {"struct1", "child1"}, then it is
    // valid if "struct1.child1" exists in this file's schema. If "struct1" exists but "child1" is
    // not a child of "struct1" then the function will return false for "struct1"
    std::function<bool(column_name_info const*, int, std::vector<column_buffer>&, bool)>
      build_column = [&](column_name_info const* col_name_info,
                         int schema_idx,
                         std::vector<column_buffer>& out_col_array,
                         bool has_list_parent) {
        if (schema_idx < 0) { return false; }
        auto const& schema_elem = get_schema(schema_idx);

        // if schema_elem is a stub then it does not exist in the column_name_info and column_buffer
        // hierarchy. So continue on
        if (schema_elem.is_stub()) {
          // is this legit?
          CUDF_EXPECTS(schema_elem.num_children == 1, "Unexpected number of children for stub");
          auto child_col_name_info = (col_name_info) ? &col_name_info->children[0] : nullptr;
          return build_column(
            child_col_name_info, schema_elem.children_idx[0], out_col_array, has_list_parent);
        }

        // if we're at the root, this is a new output column
        auto const col_type =
          schema_elem.is_one_level_list()
            ? type_id::LIST
            : to_type_id(schema_elem, strings_to_categorical, timestamp_type_id);
        auto const dtype = to_data_type(col_type, schema_elem);

        column_buffer output_col(dtype, schema_elem.repetition_type == OPTIONAL);
        if (has_list_parent) { output_col.user_data |= PARQUET_COLUMN_BUFFER_FLAG_HAS_LIST_PARENT; }
        // store the index of this element if inserted in out_col_array
        nesting.push_back(static_cast<int>(out_col_array.size()));
        output_col.name = schema_elem.name;

        // build each child
        bool path_is_valid = false;
        if (col_name_info == nullptr or col_name_info->children.empty()) {
          // add all children of schema_elem.
          // At this point, we can no longer pass a col_name_info to build_column
          for (int idx = 0; idx < schema_elem.num_children; idx++) {
            path_is_valid |= build_column(nullptr,
                                          schema_elem.children_idx[idx],
                                          output_col.children,
                                          has_list_parent || col_type == type_id::LIST);
          }
        } else {
          for (size_t idx = 0; idx < col_name_info->children.size(); idx++) {
            path_is_valid |=
              build_column(&col_name_info->children[idx],
                           find_schema_child(schema_elem, col_name_info->children[idx].name),
                           output_col.children,
                           has_list_parent || col_type == type_id::LIST);
          }
        }

        // if I have no children, we're at a leaf and I'm an input column (that is, one with actual
        // data stored) so add me to the list.
        if (schema_elem.num_children == 0) {
          input_column_info& input_col =
            input_columns.emplace_back(input_column_info{schema_idx, schema_elem.name});

          // set up child output column for one-level encoding list
          if (schema_elem.is_one_level_list()) {
            // determine the element data type
            auto const element_type =
              to_type_id(schema_elem, strings_to_categorical, timestamp_type_id);
            auto const element_dtype = to_data_type(element_type, schema_elem);

            column_buffer element_col(element_dtype, schema_elem.repetition_type == OPTIONAL);
            if (has_list_parent || col_type == type_id::LIST) {
              element_col.user_data |= PARQUET_COLUMN_BUFFER_FLAG_HAS_LIST_PARENT;
            }
            // store the index of this element
            nesting.push_back(static_cast<int>(output_col.children.size()));
            // TODO: not sure if we should assign a name or leave it blank
            element_col.name = "element";

            output_col.children.push_back(std::move(element_col));
          }

          std::copy(nesting.cbegin(), nesting.cend(), std::back_inserter(input_col.nesting));

          // pop off the extra nesting element.
          if (schema_elem.is_one_level_list()) { nesting.pop_back(); }

          path_is_valid = true;  // If we're able to reach leaf then path is valid
        }

        if (path_is_valid) { out_col_array.push_back(std::move(output_col)); }

        nesting.pop_back();
        return path_is_valid;
      };

    std::vector<int> output_column_schemas;

    //
    // there is not necessarily a 1:1 mapping between input columns and output columns.
    // For example, parquet does not explicitly store a ColumnChunkDesc for struct columns.
    // The "structiness" is simply implied by the schema.  For example, this schema:
    //  required group field_id=1 name {
    //    required binary field_id=2 firstname (String);
    //    required binary field_id=3 middlename (String);
    //    required binary field_id=4 lastname (String);
    // }
    // will only contain 3 internal columns of data (firstname, middlename, lastname).  But of
    // course "name" is ultimately the struct column we want to return.
    //
    // "firstname", "middlename" and "lastname" represent the input columns in the file that we
    // process to produce the final cudf "name" column.
    //
    // A user can ask for a single field out of the struct e.g. firstname.
    // In this case they'll pass a fully qualified name to the schema element like
    // ["name", "firstname"]
    //
    auto const& root = get_schema(0);
    if (not use_names.has_value()) {
      for (auto const& schema_idx : root.children_idx) {
        build_column(nullptr, schema_idx, output_columns, false);
        output_column_schemas.push_back(schema_idx);
      }
    } else {
      struct path_info {
        std::string full_path;
        int schema_idx;
      };

      // Convert schema into a vector of every possible path
      std::vector<path_info> all_paths;
      std::function<void(std::string, int)> add_path = [&](std::string path_till_now,
                                                           int schema_idx) {
        auto const& schema_elem = get_schema(schema_idx);
        std::string curr_path   = path_till_now + schema_elem.name;
        all_paths.push_back({curr_path, schema_idx});
        for (auto const& child_idx : schema_elem.children_idx) {
          add_path(curr_path + ".", child_idx);
        }
      };
      for (auto const& child_idx : get_schema(0).children_idx) {
        add_path("", child_idx);
      }

      // Find which of the selected paths are valid and get their schema index
      std::vector<path_info> valid_selected_paths;
      for (auto const& selected_path : *use_names) {
        auto found_path =
          std::find_if(all_paths.begin(), all_paths.end(), [&](path_info& valid_path) {
            return valid_path.full_path == selected_path;
          });
        if (found_path != all_paths.end()) {
          valid_selected_paths.push_back({selected_path, found_path->schema_idx});
        }
      }

      // Now construct paths as vector of strings for further consumption
      std::vector<std::vector<std::string>> use_names3;
      std::transform(valid_selected_paths.begin(),
                     valid_selected_paths.end(),
                     std::back_inserter(use_names3),
                     [&](path_info const& valid_path) {
                       auto schema_idx = valid_path.schema_idx;
                       std::vector<std::string> result_path;
                       do {
                         SchemaElement const& elem = get_schema(schema_idx);
                         result_path.push_back(elem.name);
                         schema_idx = elem.parent_idx;
                       } while (schema_idx > 0);
                       return std::vector<std::string>(result_path.rbegin(), result_path.rend());
                     });

      std::vector<column_name_info> selected_columns;
      if (include_index) {
        std::vector<std::string> index_names = get_pandas_index_names();
        std::transform(index_names.cbegin(),
                       index_names.cend(),
                       std::back_inserter(selected_columns),
                       [](std::string const& name) { return column_name_info(name); });
      }
      // Merge the vector use_names into a set of hierarchical column_name_info objects
      /* This is because if we have columns like this:
       *     col1
       *      / \
       *    s3   f4
       *   / \
       * f5   f6
       *
       * there may be common paths in use_names like:
       * {"col1", "s3", "f5"}, {"col1", "f4"}
       * which means we want the output to contain
       *     col1
       *      / \
       *    s3   f4
       *   /
       * f5
       *
       * rather than
       *  col1   col1
       *   |      |
       *   s3     f4
       *   |
       *   f5
       */
      for (auto const& path : use_names3) {
        auto array_to_find_in = &selected_columns;
        for (size_t depth = 0; depth < path.size(); ++depth) {
          // Check if the path exists in our selected_columns and if not, add it.
          auto const& name_to_find = path[depth];
          auto found_col           = std::find_if(
            array_to_find_in->begin(),
            array_to_find_in->end(),
            [&name_to_find](column_name_info const& col) { return col.name == name_to_find; });
          if (found_col == array_to_find_in->end()) {
            auto& col        = array_to_find_in->emplace_back(name_to_find);
            array_to_find_in = &col.children;
          } else {
            // Path exists. go down further.
            array_to_find_in = &found_col->children;
          }
        }
      }
      for (auto& col : selected_columns) {
        auto const& top_level_col_schema_idx = find_schema_child(root, col.name);
        bool valid_column = build_column(&col, top_level_col_schema_idx, output_columns, false);
        if (valid_column) output_column_schemas.push_back(top_level_col_schema_idx);
      }
    }

    return std::make_tuple(
      std::move(input_columns), std::move(output_columns), std::move(output_column_schemas));
  }
};

/**
 * @brief Generate depth remappings for repetition and definition levels.
 *
 * When dealing with columns that contain lists, we must examine incoming
 * repetition and definition level pairs to determine what range of output nesting
 * is indicated when adding new values.  This function generates the mappings of
 * the R/D levels to those start/end bounds
 *
 * @param remap Maps column schema index to the R/D remapping vectors for that column
 * @param src_col_schema The column schema to generate the new mapping for
 * @param md File metadata information
 */
void generate_depth_remappings(std::map<int, std::pair<std::vector<int>, std::vector<int>>>& remap,
                               int src_col_schema,
                               aggregate_reader_metadata const& md)
{
  // already generated for this level
  if (remap.find(src_col_schema) != remap.end()) { return; }
  auto schema   = md.get_schema(src_col_schema);
  int max_depth = md.get_output_nesting_depth(src_col_schema);

  CUDF_EXPECTS(remap.find(src_col_schema) == remap.end(),
               "Attempting to remap a schema more than once");
  auto inserted =
    remap.insert(std::pair<int, std::pair<std::vector<int>, std::vector<int>>>{src_col_schema, {}});
  auto& depth_remap = inserted.first->second;

  std::vector<int>& rep_depth_remap = (depth_remap.first);
  rep_depth_remap.resize(schema.max_repetition_level + 1);
  std::vector<int>& def_depth_remap = (depth_remap.second);
  def_depth_remap.resize(schema.max_definition_level + 1);

  // the key:
  // for incoming level values  R/D
  // add values starting at the shallowest nesting level X has repetition level R
  // until you reach the deepest nesting level Y that corresponds to the repetition level R1
  // held by the nesting level that has definition level D
  //
  // Example: a 3 level struct with a list at the bottom
  //
  //                     R / D   Depth
  // level0              0 / 1     0
  //   level1            0 / 2     1
  //     level2          0 / 3     2
  //       list          0 / 3     3
  //         element     1 / 4     4
  //
  // incoming R/D : 0, 0  -> add values from depth 0 to 3   (def level 0 always maps to depth 0)
  // incoming R/D : 0, 1  -> add values from depth 0 to 3
  // incoming R/D : 0, 2  -> add values from depth 0 to 3
  // incoming R/D : 1, 4  -> add values from depth 4 to 4
  //
  // Note : the -validity- of values is simply checked by comparing the incoming D value against the
  // D value of the given nesting level (incoming D >= the D for the nesting level == valid,
  // otherwise NULL).  The tricky part is determining what nesting levels to add values at.
  //
  // For schemas with no repetition level (no lists), X is always 0 and Y is always max nesting
  // depth.
  //

  // compute "X" from above
  for (int s_idx = schema.max_repetition_level; s_idx >= 0; s_idx--) {
    auto find_shallowest = [&](int r) {
      int shallowest = -1;
      int cur_depth  = max_depth - 1;
      int schema_idx = src_col_schema;
      while (schema_idx > 0) {
        auto cur_schema = md.get_schema(schema_idx);
        if (cur_schema.max_repetition_level == r) {
          // if this is a repeated field, map it one level deeper
          shallowest = cur_schema.is_stub() ? cur_depth + 1 : cur_depth;
        }
        // if it's one-level encoding list
        else if (cur_schema.is_one_level_list()) {
          shallowest = cur_depth - 1;
        }
        if (!cur_schema.is_stub()) { cur_depth--; }
        schema_idx = cur_schema.parent_idx;
      }
      return shallowest;
    };
    rep_depth_remap[s_idx] = find_shallowest(s_idx);
  }

  // compute "Y" from above
  for (int s_idx = schema.max_definition_level; s_idx >= 0; s_idx--) {
    auto find_deepest = [&](int d) {
      SchemaElement prev_schema;
      int schema_idx = src_col_schema;
      int r1         = 0;
      while (schema_idx > 0) {
        SchemaElement cur_schema = md.get_schema(schema_idx);
        if (cur_schema.max_definition_level == d) {
          // if this is a repeated field, map it one level deeper
          r1 = cur_schema.is_stub() ? prev_schema.max_repetition_level
                                    : cur_schema.max_repetition_level;
          break;
        }
        prev_schema = cur_schema;
        schema_idx  = cur_schema.parent_idx;
      }

      // we now know R1 from above. return the deepest nesting level that has the
      // same repetition level
      schema_idx = src_col_schema;
      int depth  = max_depth - 1;
      while (schema_idx > 0) {
        SchemaElement cur_schema = md.get_schema(schema_idx);
        if (cur_schema.max_repetition_level == r1) {
          // if this is a repeated field, map it one level deeper
          depth = cur_schema.is_stub() ? depth + 1 : depth;
          break;
        }
        if (!cur_schema.is_stub()) { depth--; }
        prev_schema = cur_schema;
        schema_idx  = cur_schema.parent_idx;
      }
      return depth;
    };
    def_depth_remap[s_idx] = find_deepest(s_idx);
  }
}

/**
 * @copydoc cudf::io::detail::parquet::read_column_chunks
 */
std::future<void> reader::impl::read_column_chunks(
  std::vector<std::unique_ptr<datasource::buffer>>& page_data,
  hostdevice_vector<gpu::ColumnChunkDesc>& chunks,  // TODO const?
  size_t begin_chunk,
  size_t end_chunk,
  const std::vector<size_t>& column_chunk_offsets,
  std::vector<size_type> const& chunk_source_map)
{
  // Transfer chunk data, coalescing adjacent chunks
  std::vector<std::future<size_t>> read_tasks;
  for (size_t chunk = begin_chunk; chunk < end_chunk;) {
    const size_t io_offset   = column_chunk_offsets[chunk];
    size_t io_size           = chunks[chunk].compressed_size;
    size_t next_chunk        = chunk + 1;
    const bool is_compressed = (chunks[chunk].codec != parquet::Compression::UNCOMPRESSED);
    while (next_chunk < end_chunk) {
      const size_t next_offset = column_chunk_offsets[next_chunk];
      const bool is_next_compressed =
        (chunks[next_chunk].codec != parquet::Compression::UNCOMPRESSED);
      if (next_offset != io_offset + io_size || is_next_compressed != is_compressed) {
        // Can't merge if not contiguous or mixing compressed and uncompressed
        // Not coalescing uncompressed with compressed chunks is so that compressed buffers can be
        // freed earlier (immediately after decompression stage) to limit peak memory requirements
        break;
      }
      io_size += chunks[next_chunk].compressed_size;
      next_chunk++;
    }
    if (io_size != 0) {
      auto& source = _sources[chunk_source_map[chunk]];
      if (source->is_device_read_preferred(io_size)) {
        auto buffer        = rmm::device_buffer(io_size, _stream);
        auto fut_read_size = source->device_read_async(
          io_offset, io_size, static_cast<uint8_t*>(buffer.data()), _stream);
        read_tasks.emplace_back(std::move(fut_read_size));
        page_data[chunk] = datasource::buffer::create(std::move(buffer));
      } else {
        auto const buffer = source->host_read(io_offset, io_size);
        page_data[chunk] =
          datasource::buffer::create(rmm::device_buffer(buffer->data(), buffer->size(), _stream));
      }
      auto d_compdata = page_data[chunk]->data();
      do {
        chunks[chunk].compressed_data = d_compdata;
        d_compdata += chunks[chunk].compressed_size;
      } while (++chunk != next_chunk);
    } else {
      chunk = next_chunk;
    }
  }
  auto sync_fn = [](decltype(read_tasks) read_tasks) {
    for (auto& task : read_tasks) {
      task.wait();
    }
  };
  return std::async(std::launch::deferred, sync_fn, std::move(read_tasks));
}

/**
 * @copydoc cudf::io::detail::parquet::count_page_headers
 */
size_t reader::impl::count_page_headers(hostdevice_vector<gpu::ColumnChunkDesc>& chunks)
{
  size_t total_pages = 0;

  chunks.host_to_device(_stream);
  gpu::DecodePageHeaders(chunks.device_ptr(), chunks.size(), _stream);
  chunks.device_to_host(_stream, true);

  for (size_t c = 0; c < chunks.size(); c++) {
    total_pages += chunks[c].num_data_pages + chunks[c].num_dict_pages;
  }

  return total_pages;
}

/**
 * @copydoc cudf::io::detail::parquet::decode_page_headers
 */
void reader::impl::decode_page_headers(hostdevice_vector<gpu::ColumnChunkDesc>& chunks,
                                       hostdevice_vector<gpu::PageInfo>& pages)
{
  // IMPORTANT : if you change how pages are stored within a chunk (dist pages, then data pages),
  // please update preprocess_nested_columns to reflect this.
  for (size_t c = 0, page_count = 0; c < chunks.size(); c++) {
    chunks[c].max_num_pages = chunks[c].num_data_pages + chunks[c].num_dict_pages;
    chunks[c].page_info     = pages.device_ptr(page_count);
    page_count += chunks[c].max_num_pages;
  }

  chunks.host_to_device(_stream);
  gpu::DecodePageHeaders(chunks.device_ptr(), chunks.size(), _stream);
  pages.device_to_host(_stream, true);
}

/**
 * @copydoc cudf::io::detail::parquet::decompress_page_data
 */
rmm::device_buffer reader::impl::decompress_page_data(
  hostdevice_vector<gpu::ColumnChunkDesc>& chunks, hostdevice_vector<gpu::PageInfo>& pages)
{
  auto for_each_codec_page = [&](parquet::Compression codec, const std::function<void(size_t)>& f) {
    for (size_t c = 0, page_count = 0; c < chunks.size(); c++) {
      const auto page_stride = chunks[c].max_num_pages;
      if (chunks[c].codec == codec) {
        for (int k = 0; k < page_stride; k++) {
          f(page_count + k);
        }
      }
      page_count += page_stride;
    }
  };

  // Brotli scratch memory for decompressing
  rmm::device_buffer debrotli_scratch;

  // Count the exact number of compressed pages
  size_t num_comp_pages    = 0;
  size_t total_decomp_size = 0;

  struct codec_stats {
    parquet::Compression compression_type = UNCOMPRESSED;
    size_t num_pages                      = 0;
    int32_t max_decompressed_size         = 0;
    size_t total_decomp_size              = 0;
  };

  std::array codecs{codec_stats{parquet::GZIP},
                    codec_stats{parquet::SNAPPY},
                    codec_stats{parquet::BROTLI},
                    codec_stats{parquet::ZSTD}};

  auto is_codec_supported = [&codecs](int8_t codec) {
    if (codec == parquet::UNCOMPRESSED) return true;
    return std::find_if(codecs.begin(), codecs.end(), [codec](auto& cstats) {
             return codec == cstats.compression_type;
           }) != codecs.end();
  };
  CUDF_EXPECTS(std::all_of(chunks.begin(),
                           chunks.end(),
                           [&is_codec_supported](auto const& chunk) {
                             return is_codec_supported(chunk.codec);
                           }),
               "Unsupported compression type");

  for (auto& codec : codecs) {
    for_each_codec_page(codec.compression_type, [&](size_t page) {
      auto page_uncomp_size = pages[page].uncompressed_page_size;
      total_decomp_size += page_uncomp_size;
      codec.total_decomp_size += page_uncomp_size;
      codec.max_decompressed_size = std::max(codec.max_decompressed_size, page_uncomp_size);
      codec.num_pages++;
      num_comp_pages++;
    });
    if (codec.compression_type == parquet::BROTLI && codec.num_pages > 0) {
      debrotli_scratch.resize(get_gpu_debrotli_scratch_size(codec.num_pages), _stream);
    }
  }

  // Dispatch batches of pages to decompress for each codec
  rmm::device_buffer decomp_pages(total_decomp_size, _stream);

  std::vector<device_span<uint8_t const>> comp_in;
  comp_in.reserve(num_comp_pages);
  std::vector<device_span<uint8_t>> comp_out;
  comp_out.reserve(num_comp_pages);

  rmm::device_uvector<decompress_status> comp_stats(num_comp_pages, _stream);
  thrust::fill(rmm::exec_policy(_stream),
               comp_stats.begin(),
               comp_stats.end(),
               decompress_status{0, static_cast<uint32_t>(-1000), 0});

  size_t decomp_offset = 0;
  int32_t start_pos    = 0;
  for (const auto& codec : codecs) {
    if (codec.num_pages == 0) { continue; }

    for_each_codec_page(codec.compression_type, [&](size_t page) {
      auto dst_base = static_cast<uint8_t*>(decomp_pages.data());
      comp_in.emplace_back(pages[page].page_data,
                           static_cast<size_t>(pages[page].compressed_page_size));
      comp_out.emplace_back(dst_base + decomp_offset,
                            static_cast<size_t>(pages[page].uncompressed_page_size));

      pages[page].page_data = static_cast<uint8_t*>(comp_out.back().data());
      decomp_offset += comp_out.back().size();
    });

    host_span<device_span<uint8_t const> const> comp_in_view{comp_in.data() + start_pos,
                                                             codec.num_pages};
    auto const d_comp_in = cudf::detail::make_device_uvector_async(comp_in_view, _stream);
    host_span<device_span<uint8_t> const> comp_out_view(comp_out.data() + start_pos,
                                                        codec.num_pages);
    auto const d_comp_out = cudf::detail::make_device_uvector_async(comp_out_view, _stream);
    device_span<decompress_status> d_comp_stats_view(comp_stats.data() + start_pos,
                                                     codec.num_pages);

    switch (codec.compression_type) {
      case parquet::GZIP:
        gpuinflate(d_comp_in, d_comp_out, d_comp_stats_view, gzip_header_included::YES, _stream);
        break;
      case parquet::SNAPPY:
        if (nvcomp_integration::is_stable_enabled()) {
          nvcomp::batched_decompress(nvcomp::compression_type::SNAPPY,
                                     d_comp_in,
                                     d_comp_out,
                                     d_comp_stats_view,
                                     codec.max_decompressed_size,
                                     codec.total_decomp_size,
                                     _stream);
        } else {
          gpu_unsnap(d_comp_in, d_comp_out, d_comp_stats_view, _stream);
        }
        break;
      case parquet::ZSTD:
        nvcomp::batched_decompress(nvcomp::compression_type::ZSTD,
                                   d_comp_in,
                                   d_comp_out,
                                   d_comp_stats_view,
                                   codec.max_decompressed_size,
                                   codec.total_decomp_size,
                                   _stream);
        break;
      case parquet::BROTLI:
        gpu_debrotli(d_comp_in,
                     d_comp_out,
                     d_comp_stats_view,
                     debrotli_scratch.data(),
                     debrotli_scratch.size(),
                     _stream);
        break;
      default: CUDF_FAIL("Unexpected decompression dispatch"); break;
    }
    start_pos += codec.num_pages;
  }

  decompress_check(comp_stats, _stream);

  // Update the page information in device memory with the updated value of
  // page_data; it now points to the uncompressed data buffer
  pages.host_to_device(_stream);

  return decomp_pages;
}

/**
 * @copydoc cudf::io::detail::parquet::allocate_nesting_info
 */
void reader::impl::allocate_nesting_info(hostdevice_vector<gpu::ColumnChunkDesc> const& chunks,
                                         hostdevice_vector<gpu::PageInfo>& pages,
                                         hostdevice_vector<gpu::PageNestingInfo>& page_nesting_info)
{
  // compute total # of page_nesting infos needed and allocate space. doing this in one
  // buffer to keep it to a single gpu allocation
  size_t const total_page_nesting_infos = std::accumulate(
    chunks.host_ptr(), chunks.host_ptr() + chunks.size(), 0, [&](int total, auto& chunk) {
      // the schema of the input column
      auto const& schema                    = _metadata->get_schema(chunk.src_col_schema);
      auto const per_page_nesting_info_size = max(
        schema.max_definition_level + 1, _metadata->get_output_nesting_depth(chunk.src_col_schema));
      return total + (per_page_nesting_info_size * chunk.num_data_pages);
    });

  page_nesting_info = hostdevice_vector<gpu::PageNestingInfo>{total_page_nesting_infos, _stream};

  // retrieve from the gpu so we can update
  pages.device_to_host(_stream, true);

  // update pointers in the PageInfos
  int target_page_index = 0;
  int src_info_index    = 0;
  for (size_t idx = 0; idx < chunks.size(); idx++) {
    int src_col_schema                    = chunks[idx].src_col_schema;
    auto& schema                          = _metadata->get_schema(src_col_schema);
    auto const per_page_nesting_info_size = std::max(
      schema.max_definition_level + 1, _metadata->get_output_nesting_depth(src_col_schema));

    // skip my dict pages
    target_page_index += chunks[idx].num_dict_pages;
    for (int p_idx = 0; p_idx < chunks[idx].num_data_pages; p_idx++) {
      pages[target_page_index + p_idx].nesting = page_nesting_info.device_ptr() + src_info_index;
      pages[target_page_index + p_idx].num_nesting_levels = per_page_nesting_info_size;

      src_info_index += per_page_nesting_info_size;
    }
    target_page_index += chunks[idx].num_data_pages;
  }

  // copy back to the gpu
  pages.host_to_device(_stream);

  // fill in
  int nesting_info_index = 0;
  std::map<int, std::pair<std::vector<int>, std::vector<int>>> depth_remapping;
  for (size_t idx = 0; idx < chunks.size(); idx++) {
    int src_col_schema = chunks[idx].src_col_schema;

    // schema of the input column
    auto& schema = _metadata->get_schema(src_col_schema);
    // real depth of the output cudf column hierarchy (1 == no nesting, 2 == 1 level, etc)
    int max_depth = _metadata->get_output_nesting_depth(src_col_schema);

    // # of nesting infos stored per page for this column
    auto const per_page_nesting_info_size = std::max(schema.max_definition_level + 1, max_depth);

    // if this column has lists, generate depth remapping
    std::map<int, std::pair<std::vector<int>, std::vector<int>>> depth_remapping;
    if (schema.max_repetition_level > 0) {
      generate_depth_remappings(depth_remapping, src_col_schema, *_metadata);
    }

    // fill in host-side nesting info
    int schema_idx  = src_col_schema;
    auto cur_schema = _metadata->get_schema(schema_idx);
    int cur_depth   = max_depth - 1;
    while (schema_idx > 0) {
      // stub columns (basically the inner field of a list scheme element) are not real columns.
      // we can ignore them for the purposes of output nesting info
      if (!cur_schema.is_stub()) {
        // initialize each page within the chunk
        for (int p_idx = 0; p_idx < chunks[idx].num_data_pages; p_idx++) {
          gpu::PageNestingInfo* pni =
            &page_nesting_info[nesting_info_index + (p_idx * per_page_nesting_info_size)];

          // if we have lists, set our start and end depth remappings
          if (schema.max_repetition_level > 0) {
            auto remap = depth_remapping.find(src_col_schema);
            CUDF_EXPECTS(remap != depth_remapping.end(),
                         "Could not find depth remapping for schema");
            std::vector<int> const& rep_depth_remap = (remap->second.first);
            std::vector<int> const& def_depth_remap = (remap->second.second);

            for (size_t m = 0; m < rep_depth_remap.size(); m++) {
              pni[m].start_depth = rep_depth_remap[m];
            }
            for (size_t m = 0; m < def_depth_remap.size(); m++) {
              pni[m].end_depth = def_depth_remap[m];
            }
          }

          // values indexed by output column index
          pni[cur_depth].max_def_level = cur_schema.max_definition_level;
          pni[cur_depth].max_rep_level = cur_schema.max_repetition_level;
          pni[cur_depth].size          = 0;
        }

        // move up the hierarchy
        cur_depth--;
      }

      // next schema
      schema_idx = cur_schema.parent_idx;
      cur_schema = _metadata->get_schema(schema_idx);
    }

    nesting_info_index += (per_page_nesting_info_size * chunks[idx].num_data_pages);
  }

  // copy nesting info to the device
  page_nesting_info.host_to_device(_stream);
}

/**
 * @copydoc cudf::io::detail::parquet::preprocess_columns
 */
void reader::impl::preprocess_columns(hostdevice_vector<gpu::ColumnChunkDesc>& chunks,
                                      hostdevice_vector<gpu::PageInfo>& pages,
                                      size_t num_rows,
                                      bool has_lists)
{
  // TODO : we should be selectively preprocessing only columns that have
  // lists in them instead of doing them all if even one contains lists.

  // if there are no lists, simply allocate every allocate every output
  // column to be of size num_rows
  if (!has_lists) {
    std::function<void(std::vector<column_buffer>&)> create_columns =
      [&](std::vector<column_buffer>& cols) {
        for (size_t idx = 0; idx < cols.size(); idx++) {
          auto& col = cols[idx];
          col.create(num_rows, _stream, _mr);
          create_columns(col.children);
        }
      };
    create_columns(_output_columns);
  } else {
    // preprocess per-nesting level sizes by page
    gpu::PreprocessColumnData(
      pages, chunks, _input_columns, _output_columns, num_rows, _stream, _mr);
    _stream.synchronize();
  }
}

/**
 * @copydoc cudf::io::detail::parquet::decode_page_data
 */
void reader::impl::decode_page_data(hostdevice_vector<gpu::ColumnChunkDesc>& chunks,
                                    hostdevice_vector<gpu::PageInfo>& pages,
                                    hostdevice_vector<gpu::PageNestingInfo>& page_nesting,
                                    size_t total_rows)
{
  auto is_dict_chunk = [](const gpu::ColumnChunkDesc& chunk) {
    return (chunk.data_type & 0x7) == BYTE_ARRAY && chunk.num_dict_pages > 0;
  };

  // Count the number of string dictionary entries
  // NOTE: Assumes first page in the chunk is always the dictionary page
  size_t total_str_dict_indexes = 0;
  for (size_t c = 0, page_count = 0; c < chunks.size(); c++) {
    if (is_dict_chunk(chunks[c])) { total_str_dict_indexes += pages[page_count].num_input_values; }
    page_count += chunks[c].max_num_pages;
  }

  // Build index for string dictionaries since they can't be indexed
  // directly due to variable-sized elements
  auto str_dict_index = cudf::detail::make_zeroed_device_uvector_async<string_index_pair>(
    total_str_dict_indexes, _stream);

  // TODO (dm): hd_vec should have begin and end iterator members
  size_t sum_max_depths =
    std::accumulate(chunks.host_ptr(),
                    chunks.host_ptr(chunks.size()),
                    0,
                    [&](size_t cursum, gpu::ColumnChunkDesc const& chunk) {
                      return cursum + _metadata->get_output_nesting_depth(chunk.src_col_schema);
                    });

  // In order to reduce the number of allocations of hostdevice_vector, we allocate a single vector
  // to store all per-chunk pointers to nested data/nullmask. `chunk_offsets[i]` will store the
  // offset into `chunk_nested_data`/`chunk_nested_valids` for the array of pointers for chunk `i`
  auto chunk_nested_valids = hostdevice_vector<uint32_t*>(sum_max_depths, _stream);
  auto chunk_nested_data   = hostdevice_vector<void*>(sum_max_depths, _stream);
  auto chunk_offsets       = std::vector<size_t>();

  // Update chunks with pointers to column data.
  for (size_t c = 0, page_count = 0, str_ofs = 0, chunk_off = 0; c < chunks.size(); c++) {
    input_column_info const& input_col = _input_columns[chunks[c].src_col_index];
    CUDF_EXPECTS(input_col.schema_idx == chunks[c].src_col_schema,
                 "Column/page schema index mismatch");

    if (is_dict_chunk(chunks[c])) {
      chunks[c].str_dict_index = str_dict_index.data() + str_ofs;
      str_ofs += pages[page_count].num_input_values;
    }

    size_t max_depth = _metadata->get_output_nesting_depth(chunks[c].src_col_schema);
    chunk_offsets.push_back(chunk_off);

    // get a slice of size `nesting depth` from `chunk_nested_valids` to store an array of pointers
    // to validity data
    auto valids              = chunk_nested_valids.host_ptr(chunk_off);
    chunks[c].valid_map_base = chunk_nested_valids.device_ptr(chunk_off);

    // get a slice of size `nesting depth` from `chunk_nested_data` to store an array of pointers to
    // out data
    auto data                  = chunk_nested_data.host_ptr(chunk_off);
    chunks[c].column_data_base = chunk_nested_data.device_ptr(chunk_off);

    chunk_off += max_depth;

    // fill in the arrays on the host.  there are some important considerations to
    // take into account here for nested columns.  specifically, with structs
    // there is sharing of output buffers between input columns.  consider this schema
    //
    //  required group field_id=1 name {
    //    required binary field_id=2 firstname (String);
    //    required binary field_id=3 middlename (String);
    //    required binary field_id=4 lastname (String);
    // }
    //
    // there are 3 input columns of data here (firstname, middlename, lastname), but
    // only 1 output column (name).  The structure of the output column buffers looks like
    // the schema itself
    //
    // struct      (name)
    //     string  (firstname)
    //     string  (middlename)
    //     string  (lastname)
    //
    // The struct column can contain validity information. the problem is, the decode
    // step for the input columns will all attempt to decode this validity information
    // because each one has it's own copy of the repetition/definition levels. but
    // since this is all happening in parallel it would mean multiple blocks would
    // be stomping all over the same memory randomly.  to work around this, we set
    // things up so that only 1 child of any given nesting level fills in the
    // data (offsets in the case of lists) or validity information for the higher
    // levels of the hierarchy that are shared.  In this case, it would mean we
    // would just choose firstname to be the one that decodes the validity for name.
    //
    // we do this by only handing out the pointers to the first child we come across.
    //
    auto* cols = &_output_columns;
    for (size_t idx = 0; idx < max_depth; idx++) {
      auto& out_buf = (*cols)[input_col.nesting[idx]];
      cols          = &out_buf.children;

      int owning_schema = out_buf.user_data & PARQUET_COLUMN_BUFFER_SCHEMA_MASK;
      if (owning_schema == 0 || owning_schema == input_col.schema_idx) {
        valids[idx] = out_buf.null_mask();
        data[idx]   = out_buf.data();
        out_buf.user_data |=
          static_cast<uint32_t>(input_col.schema_idx) & PARQUET_COLUMN_BUFFER_SCHEMA_MASK;
      } else {
        valids[idx] = nullptr;
        data[idx]   = nullptr;
      }
    }

    // column_data_base will always point to leaf data, even for nested types.
    page_count += chunks[c].max_num_pages;
  }

  chunks.host_to_device(_stream);
  chunk_nested_valids.host_to_device(_stream);
  chunk_nested_data.host_to_device(_stream);

  if (total_str_dict_indexes > 0) {
    gpu::BuildStringDictionaryIndex(chunks.device_ptr(), chunks.size(), _stream);
  }

  gpu::DecodePageData(pages, chunks, total_rows, _stream);
  pages.device_to_host(_stream);
  page_nesting.device_to_host(_stream);
  _stream.synchronize();

  // for list columns, add the final offset to every offset buffer.
  // TODO : make this happen in more efficiently. Maybe use thrust::for_each
  // on each buffer.  Or potentially do it in PreprocessColumnData
  // Note : the reason we are doing this here instead of in the decode kernel is
  // that it is difficult/impossible for a given page to know that it is writing the very
  // last value that should then be followed by a terminator (because rows can span
  // page boundaries).
  for (size_t idx = 0; idx < _input_columns.size(); idx++) {
    input_column_info const& input_col = _input_columns[idx];

    auto* cols = &_output_columns;
    for (size_t l_idx = 0; l_idx < input_col.nesting_depth(); l_idx++) {
      auto& out_buf = (*cols)[input_col.nesting[l_idx]];
      cols          = &out_buf.children;

      if (out_buf.type.id() != type_id::LIST ||
          (out_buf.user_data & PARQUET_COLUMN_BUFFER_FLAG_LIST_TERMINATED)) {
        continue;
      }
      CUDF_EXPECTS(l_idx < input_col.nesting_depth() - 1, "Encountered a leaf list column");
      auto& child = (*cols)[input_col.nesting[l_idx + 1]];

      // the final offset for a list at level N is the size of it's child
      int offset = child.type.id() == type_id::LIST ? child.size - 1 : child.size;
      hipMemcpyAsync(static_cast<int32_t*>(out_buf.data()) + (out_buf.size - 1),
                      &offset,
                      sizeof(offset),
                      hipMemcpyHostToDevice,
                      _stream.value());
      out_buf.user_data |= PARQUET_COLUMN_BUFFER_FLAG_LIST_TERMINATED;
    }
  }

  // update null counts in the final column buffers
  for (size_t idx = 0; idx < pages.size(); idx++) {
    gpu::PageInfo* pi = &pages[idx];
    if (pi->flags & gpu::PAGEINFO_FLAGS_DICTIONARY) { continue; }
    gpu::ColumnChunkDesc* col          = &chunks[pi->chunk_idx];
    input_column_info const& input_col = _input_columns[col->src_col_index];

    int index                 = pi->nesting - page_nesting.device_ptr();
    gpu::PageNestingInfo* pni = &page_nesting[index];

    auto* cols = &_output_columns;
    for (size_t l_idx = 0; l_idx < input_col.nesting_depth(); l_idx++) {
      auto& out_buf = (*cols)[input_col.nesting[l_idx]];
      cols          = &out_buf.children;

      // if I wasn't the one who wrote out the validity bits, skip it
      if (chunk_nested_valids.host_ptr(chunk_offsets[pi->chunk_idx])[l_idx] == nullptr) {
        continue;
      }
      out_buf.null_count() += pni[l_idx].null_count;
    }
  }

  _stream.synchronize();
}

reader::impl::impl(std::vector<std::unique_ptr<datasource>>&& sources,
                   parquet_reader_options const& options,
                   rmm::cuda_stream_view stream,
                   rmm::mr::device_memory_resource* mr)
  : _stream(stream), _mr(mr), _sources(std::move(sources))
{
  // Open and parse the source dataset metadata
  _metadata = std::make_unique<aggregate_reader_metadata>(_sources);

  // Override output timestamp resolution if requested
  if (options.get_timestamp_type().id() != type_id::EMPTY) {
    _timestamp_type = options.get_timestamp_type();
  }

  // Strings may be returned as either string or categorical columns
  _strings_to_categorical = options.is_enabled_convert_strings_to_categories();

  // Binary columns can be read as binary or strings
  _reader_column_schema = options.get_column_schema();

  // Select only columns required by the options
  std::tie(_input_columns, _output_columns, _output_column_schemas) =
    _metadata->select_columns(options.get_columns(),
                              options.is_enabled_use_pandas_metadata(),
                              _strings_to_categorical,
                              _timestamp_type.id());
}

table_with_metadata reader::impl::read(std::vector<std::vector<size_type>> const& row_group_list)
{
  // Select only row groups required
  const auto [selected_row_groups, num_rows] = _metadata->select_row_groups(row_group_list);

  table_metadata out_metadata;

  // output cudf columns as determined by the top level schema
  std::vector<std::unique_ptr<column>> out_columns;
  out_columns.reserve(_output_columns.size());

  if (selected_row_groups.size() != 0 && _input_columns.size() != 0) {
    // Descriptors for all the chunks that make up the selected columns
    const auto num_input_columns = _input_columns.size();
    const auto num_chunks        = selected_row_groups.size() * num_input_columns;
    hostdevice_vector<gpu::ColumnChunkDesc> chunks(0, num_chunks, _stream);

    // Association between each column chunk and its source
    std::vector<size_type> chunk_source_map(num_chunks);

    // Tracker for eventually deallocating compressed and uncompressed data
    std::vector<std::unique_ptr<datasource::buffer>> page_data(num_chunks);

    // Keep track of column chunk file offsets
    std::vector<size_t> column_chunk_offsets(num_chunks);

    // if there are lists present, we need to preprocess
    bool has_lists = false;

    // Initialize column chunk information
    size_t total_decompressed_size = 0;
    auto remaining_rows            = num_rows;
    std::vector<std::future<void>> read_rowgroup_tasks;
    for (const auto& rg : selected_row_groups) {
      const auto& row_group       = _metadata->get_row_group(rg.index, rg.source_index);
      auto const row_group_start  = rg.start_row;
      auto const row_group_source = rg.source_index;
      auto const row_group_rows   = std::min<int>(remaining_rows, row_group.num_rows);
      auto const io_chunk_idx     = chunks.size();

      // generate ColumnChunkDesc objects for everything to be decoded (all input columns)
      for (size_t i = 0; i < num_input_columns; ++i) {
        auto col = _input_columns[i];
        // look up metadata
        auto& col_meta = _metadata->get_column_metadata(rg.index, rg.source_index, col.schema_idx);
        auto& schema   = _metadata->get_schema(col.schema_idx);

        // this column contains repetition levels and will require a preprocess
        if (schema.max_repetition_level > 0) { has_lists = true; }

        // Spec requires each row group to contain exactly one chunk for every
        // column. If there are too many or too few, continue with best effort
        if (chunks.size() >= chunks.max_size()) {
          std::cerr << "Detected too many column chunks" << std::endl;
          continue;
        }

        auto [type_width, clock_rate, converted_type] =
          conversion_info(to_type_id(schema, _strings_to_categorical, _timestamp_type.id()),
                          _timestamp_type.id(),
                          schema.type,
                          schema.converted_type,
                          schema.type_length);

        column_chunk_offsets[chunks.size()] =
          (col_meta.dictionary_page_offset != 0)
            ? std::min(col_meta.data_page_offset, col_meta.dictionary_page_offset)
            : col_meta.data_page_offset;

        chunks.insert(gpu::ColumnChunkDesc(col_meta.total_compressed_size,
                                           nullptr,
                                           col_meta.num_values,
                                           schema.type,
                                           type_width,
                                           row_group_start,
                                           row_group_rows,
                                           schema.max_definition_level,
                                           schema.max_repetition_level,
                                           _metadata->get_output_nesting_depth(col.schema_idx),
                                           required_bits(schema.max_definition_level),
                                           required_bits(schema.max_repetition_level),
                                           col_meta.codec,
                                           converted_type,
                                           schema.logical_type,
                                           schema.decimal_scale,
                                           clock_rate,
                                           i,
                                           col.schema_idx));

        // Map each column chunk to its column index and its source index
        chunk_source_map[chunks.size() - 1] = row_group_source;

        if (col_meta.codec != Compression::UNCOMPRESSED) {
          total_decompressed_size += col_meta.total_uncompressed_size;
        }
      }
      // Read compressed chunk data to device memory
      read_rowgroup_tasks.push_back(read_column_chunks(
        page_data, chunks, io_chunk_idx, chunks.size(), column_chunk_offsets, chunk_source_map));

      remaining_rows -= row_group.num_rows;
    }
    for (auto& task : read_rowgroup_tasks) {
      task.wait();
    }
    assert(remaining_rows <= 0);

    // Process dataset chunk pages into output columns
    const auto total_pages = count_page_headers(chunks);
    if (total_pages > 0) {
      hostdevice_vector<gpu::PageInfo> pages(total_pages, total_pages, _stream);
      rmm::device_buffer decomp_page_data;

      // decoding of column/page information
      decode_page_headers(chunks, pages);
      if (total_decompressed_size > 0) {
        decomp_page_data = decompress_page_data(chunks, pages);
        // Free compressed data
        for (size_t c = 0; c < chunks.size(); c++) {
          if (chunks[c].codec != parquet::Compression::UNCOMPRESSED) { page_data[c].reset(); }
        }
      }

      // build output column info
      // walk the schema, building out_buffers that mirror what our final cudf columns will look
      // like. important : there is not necessarily a 1:1 mapping between input columns and output
      // columns. For example, parquet does not explicitly store a ColumnChunkDesc for struct
      // columns. The "structiness" is simply implied by the schema.  For example, this schema:
      //  required group field_id=1 name {
      //    required binary field_id=2 firstname (String);
      //    required binary field_id=3 middlename (String);
      //    required binary field_id=4 lastname (String);
      // }
      // will only contain 3 columns of data (firstname, middlename, lastname).  But of course
      // "name" is a struct column that we want to return, so we have to make sure that we
      // create it ourselves.
      // std::vector<output_column_info> output_info = build_output_column_info();

      // nesting information (sizes, etc) stored -per page-
      // note : even for flat schemas, we allocate 1 level of "nesting" info
      hostdevice_vector<gpu::PageNestingInfo> page_nesting_info;
      allocate_nesting_info(chunks, pages, page_nesting_info);

      // - compute column sizes and allocate output buffers.
      //   important:
      //   for nested schemas, we have to do some further preprocessing to determine:
      //    - real column output sizes per level of nesting (in a flat schema, there's only 1 level
      //    of
      //      nesting and it's size is the row count)
      //
      // - for nested schemas, output buffer offset values per-page, per nesting-level for the
      // purposes of decoding.
      preprocess_columns(chunks, pages, num_rows, has_lists);

      // decoding of column data itself
      decode_page_data(chunks, pages, page_nesting_info, num_rows);

      // create the final output cudf columns
      for (size_t i = 0; i < _output_columns.size(); ++i) {
        column_name_info& col_name = out_metadata.schema_info.emplace_back("");
        auto const metadata =
          _reader_column_schema.has_value()
            ? std::make_optional<reader_column_schema>((*_reader_column_schema)[i])
            : std::nullopt;
        out_columns.emplace_back(
          make_column(_output_columns[i], &col_name, metadata, _stream, _mr));
      }
    }
  }

  // Create empty columns as needed (this can happen if we've ended up with no actual data to read)
  for (size_t i = out_columns.size(); i < _output_columns.size(); ++i) {
    column_name_info& col_name = out_metadata.schema_info.emplace_back("");
    out_columns.emplace_back(io::detail::empty_like(_output_columns[i], &col_name, _stream, _mr));
  }

  // Return column names (must match order of returned columns)
  out_metadata.column_names.resize(_output_columns.size());
  for (size_t i = 0; i < _output_column_schemas.size(); i++) {
    auto const& schema           = _metadata->get_schema(_output_column_schemas[i]);
    out_metadata.column_names[i] = schema.name;
  }

  // Return user metadata
  out_metadata.per_file_user_data = _metadata->get_key_value_metadata();
  out_metadata.user_data          = {out_metadata.per_file_user_data[0].begin(),
                            out_metadata.per_file_user_data[0].end()};

  return {std::make_unique<table>(std::move(out_columns)), std::move(out_metadata)};
}

// Forward to implementation
reader::reader(std::vector<std::unique_ptr<cudf::io::datasource>>&& sources,
               parquet_reader_options const& options,
               rmm::cuda_stream_view stream,
               rmm::mr::device_memory_resource* mr)
  : _impl(std::make_unique<impl>(std::move(sources), options, stream, mr))
{
}

// Destructor within this translation unit
reader::~reader() = default;

// Forward to implementation
table_with_metadata reader::read(parquet_reader_options const& options)
{
  return _impl->read(options.get_row_groups());
}

}  // namespace parquet
}  // namespace detail
}  // namespace io
}  // namespace cudf
