#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019-2021, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

/**
 * @file reader_impl.cu
 * @brief cuDF-IO ORC reader class implementation
 */

#include "io/orc/orc_gpu.h"
#include "reader_impl.hpp"
#include "timezone.cuh"

#include <io/comp/gpuinflate.h>
#include "orc.h"

#include <cudf/detail/utilities/vector_factories.hpp>
#include <cudf/table/table.hpp>
#include <cudf/utilities/error.hpp>
#include <cudf/utilities/traits.hpp>

#include <iterator>
#include <rmm/cuda_stream_view.hpp>
#include <rmm/device_buffer.hpp>
#include <rmm/device_uvector.hpp>
#include <rmm/exec_policy.hpp>

#include <algorithm>
#include <array>

namespace cudf {
namespace io {
namespace detail {
namespace orc {
// Import functionality that's independent of legacy code
using namespace cudf::io::orc;
using namespace cudf::io;

namespace {
/**
 * @brief Function that translates ORC data kind to cuDF type enum
 */
constexpr type_id to_type_id(const orc::SchemaType& schema,
                             bool use_np_dtypes,
                             type_id timestamp_type_id,
                             bool decimals_as_float64)
{
  switch (schema.kind) {
    case orc::BOOLEAN: return type_id::BOOL8;
    case orc::BYTE: return type_id::INT8;
    case orc::SHORT: return type_id::INT16;
    case orc::INT: return type_id::INT32;
    case orc::LONG: return type_id::INT64;
    case orc::FLOAT: return type_id::FLOAT32;
    case orc::DOUBLE: return type_id::FLOAT64;
    case orc::STRING:
    case orc::BINARY:
    case orc::VARCHAR:
    case orc::CHAR:
      // Variable-length types can all be mapped to STRING
      return type_id::STRING;
    case orc::TIMESTAMP:
      return (timestamp_type_id != type_id::EMPTY) ? timestamp_type_id
                                                   : type_id::TIMESTAMP_NANOSECONDS;
    case orc::DATE:
      // There isn't a (DAYS -> np.dtype) mapping
      return (use_np_dtypes) ? type_id::TIMESTAMP_MILLISECONDS : type_id::TIMESTAMP_DAYS;
    case orc::DECIMAL: return (decimals_as_float64) ? type_id::FLOAT64 : type_id::DECIMAL64;
    case orc::LIST: return type_id::LIST;
    case orc::STRUCT: return type_id::STRUCT;
    default: break;
  }

  return type_id::EMPTY;
}

/**
 * @brief Function that translates cuDF time unit to ORC clock frequency
 */
constexpr int32_t to_clockrate(type_id timestamp_type_id)
{
  switch (timestamp_type_id) {
    case type_id::TIMESTAMP_SECONDS: return 1;
    case type_id::TIMESTAMP_MILLISECONDS: return 1000;
    case type_id::TIMESTAMP_MICROSECONDS: return 1000000;
    case type_id::TIMESTAMP_NANOSECONDS: return 1000000000;
    default: return 0;
  }
}

constexpr std::pair<gpu::StreamIndexType, uint32_t> get_index_type_and_pos(
  const orc::StreamKind kind, uint32_t skip_count, bool non_child)
{
  switch (kind) {
    case orc::DATA:
      skip_count += 1;
      skip_count |= (skip_count & 0xff) << 8;
      return std::make_pair(gpu::CI_DATA, skip_count);
    case orc::LENGTH:
    case orc::SECONDARY:
      skip_count += 1;
      skip_count |= (skip_count & 0xff) << 16;
      return std::make_pair(gpu::CI_DATA2, skip_count);
    case orc::DICTIONARY_DATA: return std::make_pair(gpu::CI_DICTIONARY, skip_count);
    case orc::PRESENT:
      skip_count += (non_child ? 1 : 0);
      return std::make_pair(gpu::CI_PRESENT, skip_count);
    case orc::ROW_INDEX: return std::make_pair(gpu::CI_INDEX, skip_count);
    default:
      // Skip this stream as it's not strictly required
      return std::make_pair(gpu::CI_NUM_STREAMS, 0);
  }
}

}  // namespace

namespace {
/**
 * @brief struct to store buffer data and size of list buffer
 */
struct list_buffer_data {
  size_type* data;
  size_type size;
};

// Generates offsets for list buffer from number of elements in a row.
void generate_offsets_for_list(rmm::device_uvector<list_buffer_data> const& buff_data,
                               rmm::cuda_stream_view stream)
{
  auto transformer = [] __device__(list_buffer_data list_data) {
    thrust::exclusive_scan(
      thrust::seq, list_data.data, list_data.data + list_data.size, list_data.data);
  };
  thrust::for_each(rmm::exec_policy(stream), buff_data.begin(), buff_data.end(), transformer);
  stream.synchronize();
}

/**
 * @brief Struct that maps ORC streams to columns
 */
struct orc_stream_info {
  orc_stream_info() = default;
  explicit orc_stream_info(
    uint64_t offset_, size_t dst_pos_, uint32_t length_, uint32_t gdf_idx_, uint32_t stripe_idx_)
    : offset(offset_),
      dst_pos(dst_pos_),
      length(length_),
      gdf_idx(gdf_idx_),
      stripe_idx(stripe_idx_)
  {
  }
  uint64_t offset;      // offset in file
  size_t dst_pos;       // offset in memory relative to start of compressed stripe data
  size_t length;        // length in file
  uint32_t gdf_idx;     // column index
  uint32_t stripe_idx;  // stripe index
};

/**
 * @brief Function that populates column descriptors stream/chunk
 */
size_t gather_stream_info(const size_t stripe_index,
                          const orc::StripeInformation* stripeinfo,
                          const orc::StripeFooter* stripefooter,
                          const std::vector<int>& orc2gdf,
                          const std::vector<orc_column_meta>& gdf2orc,
                          const std::vector<orc::SchemaType> types,
                          bool use_index,
                          size_t* num_dictionary_entries,
                          cudf::detail::hostdevice_2dvector<gpu::ColumnDesc>& chunks,
                          std::vector<orc_stream_info>& stream_info)
{
  uint64_t src_offset = 0;
  uint64_t dst_offset = 0;
  for (const auto& stream : stripefooter->streams) {
    if (!stream.column_id || *stream.column_id >= orc2gdf.size()) {
      dst_offset += stream.length;
      continue;
    }

    auto const column_id = *stream.column_id;
    auto col             = orc2gdf[column_id];

    if (col == -1) {
      // A struct-type column has no data itself, but rather child columns
      // for each of its fields. There is only a PRESENT stream, which
      // needs to be included for the reader.
      const auto schema_type = types[column_id];
      if (schema_type.subtypes.size() != 0) {
        if (schema_type.kind == orc::STRUCT && stream.kind == orc::PRESENT) {
          for (const auto& idx : schema_type.subtypes) {
            auto child_idx = (idx < orc2gdf.size()) ? orc2gdf[idx] : -1;
            if (child_idx >= 0) {
              col                             = child_idx;
              auto& chunk                     = chunks[stripe_index][col];
              chunk.strm_id[gpu::CI_PRESENT]  = stream_info.size();
              chunk.strm_len[gpu::CI_PRESENT] = stream.length;
            }
          }
        }
      }
    }
    if (col != -1) {
      if (src_offset >= stripeinfo->indexLength || use_index) {
        // NOTE: skip_count field is temporarily used to track index ordering
        auto& chunk = chunks[stripe_index][col];
        const auto idx =
          get_index_type_and_pos(stream.kind, chunk.skip_count, col == orc2gdf[column_id]);
        if (idx.first < gpu::CI_NUM_STREAMS) {
          chunk.strm_id[idx.first]  = stream_info.size();
          chunk.strm_len[idx.first] = stream.length;
          chunk.skip_count          = idx.second;

          if (idx.first == gpu::CI_DICTIONARY) {
            chunk.dictionary_start = *num_dictionary_entries;
            chunk.dict_len         = stripefooter->columns[column_id].dictionarySize;
            *num_dictionary_entries += stripefooter->columns[column_id].dictionarySize;
          }
        }
      }
      stream_info.emplace_back(
        stripeinfo->offset + src_offset, dst_offset, stream.length, col, stripe_index);
      dst_offset += stream.length;
    }
    src_offset += stream.length;
  }

  return dst_offset;
}

/**
 * @brief Determines if a column should be converted from decimal to float
 */
bool should_convert_decimal_column_to_float(const std::vector<std::string>& columns_to_convert,
                                            cudf::io::orc::metadata& metadata,
                                            int column_index)
{
  return (std::find(columns_to_convert.begin(),
                    columns_to_convert.end(),
                    metadata.get_column_name(column_index)) != columns_to_convert.end());
}

}  // namespace

/**
 * @brief In order to support multiple input files/buffers we need to gather
 * the metadata across all of those input(s). This class provides a place
 * to aggregate that metadata from all the files.
 */
class aggregate_orc_metadata {
  using OrcStripeInfo = std::pair<const StripeInformation*, const StripeFooter*>;

 public:
  mutable std::vector<cudf::io::orc::metadata> per_file_metadata;
  size_type const num_rows;
  size_type const num_columns;
  size_type const num_stripes;

  /**
   * @brief Create a metadata object from each element in the source vector
   */
  auto metadatas_from_sources(std::vector<std::unique_ptr<datasource>> const& sources)
  {
    std::vector<cudf::io::orc::metadata> metadatas;
    std::transform(
      sources.cbegin(), sources.cend(), std::back_inserter(metadatas), [](auto const& source) {
        return cudf::io::orc::metadata(source.get());
      });
    return metadatas;
  }

  /**
   * @brief Sums up the number of rows of each source
   */
  size_type calc_num_rows() const
  {
    return std::accumulate(
      per_file_metadata.begin(), per_file_metadata.end(), 0, [](auto& sum, auto& pfm) {
        return sum + pfm.get_total_rows();
      });
  }

  /**
   * @brief Number of columns in a ORC file.
   */
  size_type calc_num_cols() const
  {
    if (not per_file_metadata.empty()) { return per_file_metadata[0].get_num_columns(); }
    return 0;
  }

  /**
   * @brief Sums up the number of stripes of each source
   */
  size_type calc_num_stripes() const
  {
    return std::accumulate(
      per_file_metadata.begin(), per_file_metadata.end(), 0, [](auto& sum, auto& pfm) {
        return sum + pfm.get_num_stripes();
      });
  }

  aggregate_orc_metadata(std::vector<std::unique_ptr<datasource>> const& sources)
    : per_file_metadata(metadatas_from_sources(sources)),
      num_rows(calc_num_rows()),
      num_columns(calc_num_cols()),
      num_stripes(calc_num_stripes())
  {
    // Verify that the input files have the same number of columns,
    // as well as matching types, compression, and names
    for (auto const& pfm : per_file_metadata) {
      CUDF_EXPECTS(per_file_metadata[0].get_num_columns() == pfm.get_num_columns(),
                   "All sources must have the same number of columns");
      CUDF_EXPECTS(per_file_metadata[0].ps.compression == pfm.ps.compression,
                   "All sources must have the same compression type");

      // Check the types, column names, and decimal scale
      for (size_t i = 0; i < pfm.ff.types.size(); i++) {
        CUDF_EXPECTS(pfm.ff.types[i].kind == per_file_metadata[0].ff.types[i].kind,
                     "Column types across all input sources must be the same");
        CUDF_EXPECTS(std::equal(pfm.ff.types[i].fieldNames.begin(),
                                pfm.ff.types[i].fieldNames.end(),
                                per_file_metadata[0].ff.types[i].fieldNames.begin()),
                     "All source column names must be the same");
        CUDF_EXPECTS(
          pfm.ff.types[i].scale.value_or(0) == per_file_metadata[0].ff.types[i].scale.value_or(0),
          "All scale values must be the same");
      }
    }
  }

  auto const& get_schema(int schema_idx) const { return per_file_metadata[0].ff.types[schema_idx]; }

  auto get_col_type(int col_idx) const { return per_file_metadata[0].ff.types[col_idx]; }

  auto get_num_rows() const { return num_rows; }

  auto get_num_cols() const { return per_file_metadata[0].get_num_columns(); }

  auto get_num_stripes() const { return num_stripes; }

  auto get_num_source_files() const { return per_file_metadata.size(); }

  auto const& get_types() const { return per_file_metadata[0].ff.types; }

  int get_row_index_stride() const { return per_file_metadata[0].ff.rowIndexStride; }

  auto get_column_name(const int source_idx, const int column_idx) const
  {
    CUDF_EXPECTS(source_idx <= static_cast<int>(per_file_metadata.size()),
                 "Out of range source_idx provided");
    CUDF_EXPECTS(column_idx <= per_file_metadata[source_idx].get_num_columns(),
                 "Out of range column_idx provided");
    return per_file_metadata[source_idx].get_column_name(column_idx);
  }

  std::vector<cudf::io::orc::metadata::stripe_source_mapping> select_stripes(
    std::vector<std::vector<size_type>> const& user_specified_stripes,
    size_type& row_start,
    size_type& row_count)
  {
    std::vector<cudf::io::orc::metadata::stripe_source_mapping> selected_stripes_mapping;

    if (!user_specified_stripes.empty()) {
      CUDF_EXPECTS(user_specified_stripes.size() == get_num_source_files(),
                   "Must specify stripes for each source");
      // row_start is 0 if stripes are set. If this is not true anymore, then
      // row_start needs to be subtracted to get the correct row_count
      CUDF_EXPECTS(row_start == 0, "Start row index should be 0");

      row_count = 0;
      // Each vector entry represents a source file; each nested vector represents the
      // user_defined_stripes to get from that source file
      for (size_t src_file_idx = 0; src_file_idx < user_specified_stripes.size(); ++src_file_idx) {
        std::vector<OrcStripeInfo> stripe_infos;

        // Coalesce stripe info at the source file later since that makes downstream processing much
        // easier in impl::read
        for (const size_t& stripe_idx : user_specified_stripes[src_file_idx]) {
          CUDF_EXPECTS(stripe_idx < per_file_metadata[src_file_idx].ff.stripes.size(),
                       "Invalid stripe index");
          stripe_infos.push_back(
            std::make_pair(&per_file_metadata[src_file_idx].ff.stripes[stripe_idx], nullptr));
          row_count += per_file_metadata[src_file_idx].ff.stripes[stripe_idx].numberOfRows;
        }
        selected_stripes_mapping.push_back({static_cast<int>(src_file_idx), stripe_infos});
      }
    } else {
      row_start = std::max(row_start, 0);
      if (row_count < 0) {
        row_count = static_cast<size_type>(
          std::min<int64_t>(get_num_rows(), std::numeric_limits<size_type>::max()));
      }
      row_count = std::min(row_count, get_num_rows() - row_start);
      CUDF_EXPECTS(row_count >= 0, "Invalid row count");
      CUDF_EXPECTS(row_start <= get_num_rows(), "Invalid row start");

      size_type count = 0;
      // Iterate all source files, each source file has corelating metadata
      for (size_t src_file_idx = 0;
           src_file_idx < per_file_metadata.size() && count < row_start + row_count;
           ++src_file_idx) {
        std::vector<OrcStripeInfo> stripe_infos;

        for (size_t stripe_idx = 0;
             stripe_idx < per_file_metadata[src_file_idx].ff.stripes.size() &&
             count < row_start + row_count;
             ++stripe_idx) {
          count += per_file_metadata[src_file_idx].ff.stripes[stripe_idx].numberOfRows;
          if (count > row_start || count == 0) {
            stripe_infos.push_back(
              std::make_pair(&per_file_metadata[src_file_idx].ff.stripes[stripe_idx], nullptr));
          }
        }

        selected_stripes_mapping.push_back({static_cast<int>(src_file_idx), stripe_infos});
      }
    }

    // Read each stripe's stripefooter metadata
    if (not selected_stripes_mapping.empty()) {
      for (auto& mapping : selected_stripes_mapping) {
        // Resize to all stripe_info for the source level
        per_file_metadata[mapping.source_idx].stripefooters.resize(mapping.stripe_info.size());

        for (size_t i = 0; i < mapping.stripe_info.size(); i++) {
          const auto stripe         = mapping.stripe_info[i].first;
          const auto sf_comp_offset = stripe->offset + stripe->indexLength + stripe->dataLength;
          const auto sf_comp_length = stripe->footerLength;
          CUDF_EXPECTS(
            sf_comp_offset + sf_comp_length < per_file_metadata[mapping.source_idx].source->size(),
            "Invalid stripe information");
          const auto buffer =
            per_file_metadata[mapping.source_idx].source->host_read(sf_comp_offset, sf_comp_length);
          size_t sf_length = 0;
          auto sf_data     = per_file_metadata[mapping.source_idx].decompressor->Decompress(
            buffer->data(), sf_comp_length, &sf_length);
          ProtobufReader(sf_data, sf_length)
            .read(per_file_metadata[mapping.source_idx].stripefooters[i]);
          mapping.stripe_info[i].second = &per_file_metadata[mapping.source_idx].stripefooters[i];
        }
      }
    }

    return selected_stripes_mapping;
  }

  /**
   * @brief Adds column as per the request and saves metadata about children.
   *        Struct children are in the same level as struct, only list column
   *        children are pushed to next level.
   *
   * @param selection A vector that saves list of columns as per levels of nesting.
   * @param types A vector of schema types of columns.
   * @param level current level of nesting.
   * @param id current column id that needs to be added.
   * @param has_timestamp_column True if timestamp column present and false otherwise.
   *
   * @return returns number of child columns at same level in case of struct and next level in case
   * of list
   */
  uint32_t add_column(std::vector<std::vector<orc_column_meta>>& selection,
                      std::vector<SchemaType> const& types,
                      const size_t level,
                      const uint32_t id,
                      bool& has_timestamp_column,
                      bool& has_list_column)
  {
    uint32_t num_lvl_child_columns = 0;
    if (level == selection.size()) { selection.emplace_back(); }
    selection[level].push_back({id, 0});
    const int col_id = selection[level].size() - 1;
    if (types[id].kind == orc::TIMESTAMP) { has_timestamp_column = true; }

    switch (types[id].kind) {
      case orc::LIST: {
        uint32_t lvl_cols = 0;
        if (not types[id].subtypes.empty()) {
          has_list_column = true;
          // Since list column needs to be processed before its child can be processed,
          // child column is being added to next level
          lvl_cols =
            add_column(selection, types, level + 1, id + 1, has_timestamp_column, has_list_column);
        }
        // The list child column may be a struct in which case lvl_cols will be > 1
        selection[level][col_id].num_children = lvl_cols;
      } break;

      case orc::STRUCT:
        for (const auto child_id : types[id].subtypes) {
          num_lvl_child_columns +=
            add_column(selection, types, level, child_id, has_timestamp_column, has_list_column);
        }
        selection[level][col_id].num_children = num_lvl_child_columns;
        break;

      default: break;
    }

    return num_lvl_child_columns + 1;
  }

  /**
   * @brief Filters and reduces down to a selection of columns
   *
   * @param use_names List of column names to select
   * @param has_timestamp_column True if timestamp column present and false otherwise
   *
   * @return Vector of list of ORC column meta-data
   */
  std::vector<std::vector<orc_column_meta>> select_columns(
    std::vector<std::string> const& use_names, bool& has_timestamp_column, bool& has_list_column)
  {
    auto const& pfm = per_file_metadata[0];
    std::vector<std::vector<orc_column_meta>> selection;

    if (not use_names.empty()) {
      uint32_t index = 0;
      // Have to check only parent columns
      auto const num_columns = pfm.ff.types[0].subtypes.size();

      for (const auto& use_name : use_names) {
        bool name_found = false;
        for (uint32_t i = 0; i < num_columns; ++i, ++index) {
          if (index >= num_columns) { index = 0; }
          auto col_id = pfm.ff.types[0].subtypes[index];
          if (pfm.get_column_name(col_id) == use_name) {
            name_found = true;
            add_column(selection, pfm.ff.types, 0, col_id, has_timestamp_column, has_list_column);
            // Should start with next index
            index = i + 1;
            break;
          }
        }
        CUDF_EXPECTS(name_found, "Unknown column name : " + std::string(use_name));
      }
    } else {
      for (auto const& col_id : pfm.ff.types[0].subtypes) {
        add_column(selection, pfm.ff.types, 0, col_id, has_timestamp_column, has_list_column);
      }
    }

    return selection;
  }
};

rmm::device_buffer reader::impl::decompress_stripe_data(
  cudf::detail::hostdevice_2dvector<gpu::ColumnDesc>& chunks,
  const std::vector<rmm::device_buffer>& stripe_data,
  const OrcDecompressor* decompressor,
  std::vector<orc_stream_info>& stream_info,
  size_t num_stripes,
  cudf::detail::hostdevice_2dvector<gpu::RowGroup>& row_groups,
  size_t row_index_stride,
  bool use_base_stride,
  rmm::cuda_stream_view stream)
{
  // Parse the columns' compressed info
  hostdevice_vector<gpu::CompressedStreamInfo> compinfo(0, stream_info.size(), stream);
  for (const auto& info : stream_info) {
    compinfo.insert(gpu::CompressedStreamInfo(
      static_cast<const uint8_t*>(stripe_data[info.stripe_idx].data()) + info.dst_pos,
      info.length));
  }
  compinfo.host_to_device(stream);
  gpu::ParseCompressedStripeData(compinfo.device_ptr(),
                                 compinfo.size(),
                                 decompressor->GetBlockSize(),
                                 decompressor->GetLog2MaxCompressionRatio(),
                                 stream);
  compinfo.device_to_host(stream, true);

  // Count the exact number of compressed blocks
  size_t num_compressed_blocks   = 0;
  size_t num_uncompressed_blocks = 0;
  size_t total_decomp_size       = 0;
  for (size_t i = 0; i < compinfo.size(); ++i) {
    num_compressed_blocks += compinfo[i].num_compressed_blocks;
    num_uncompressed_blocks += compinfo[i].num_uncompressed_blocks;
    total_decomp_size += compinfo[i].max_uncompressed_size;
  }
  CUDF_EXPECTS(total_decomp_size > 0, "No decompressible data found");

  rmm::device_buffer decomp_data(total_decomp_size, stream);
  rmm::device_uvector<gpu_inflate_input_s> inflate_in(
    num_compressed_blocks + num_uncompressed_blocks, stream);
  rmm::device_uvector<gpu_inflate_status_s> inflate_out(num_compressed_blocks, stream);

  // Parse again to populate the decompression input/output buffers
  size_t decomp_offset      = 0;
  uint32_t start_pos        = 0;
  uint32_t start_pos_uncomp = (uint32_t)num_compressed_blocks;
  for (size_t i = 0; i < compinfo.size(); ++i) {
    auto dst_base                 = static_cast<uint8_t*>(decomp_data.data());
    compinfo[i].uncompressed_data = dst_base + decomp_offset;
    compinfo[i].decctl            = inflate_in.data() + start_pos;
    compinfo[i].decstatus         = inflate_out.data() + start_pos;
    compinfo[i].copyctl           = inflate_in.data() + start_pos_uncomp;

    stream_info[i].dst_pos = decomp_offset;
    decomp_offset += compinfo[i].max_uncompressed_size;
    start_pos += compinfo[i].num_compressed_blocks;
    start_pos_uncomp += compinfo[i].num_uncompressed_blocks;
  }
  compinfo.host_to_device(stream);
  gpu::ParseCompressedStripeData(compinfo.device_ptr(),
                                 compinfo.size(),
                                 decompressor->GetBlockSize(),
                                 decompressor->GetLog2MaxCompressionRatio(),
                                 stream);

  // Dispatch batches of blocks to decompress
  if (num_compressed_blocks > 0) {
    switch (decompressor->GetKind()) {
      case orc::ZLIB:
        CUDA_TRY(
          gpuinflate(inflate_in.data(), inflate_out.data(), num_compressed_blocks, 0, stream));
        break;
      case orc::SNAPPY:
        CUDA_TRY(gpu_unsnap(inflate_in.data(), inflate_out.data(), num_compressed_blocks, stream));
        break;
      default: CUDF_EXPECTS(false, "Unexpected decompression dispatch"); break;
    }
  }
  if (num_uncompressed_blocks > 0) {
    CUDA_TRY(gpu_copy_uncompressed_blocks(
      inflate_in.data() + num_compressed_blocks, num_uncompressed_blocks, stream));
  }
  gpu::PostDecompressionReassemble(compinfo.device_ptr(), compinfo.size(), stream);

  // Update the stream information with the updated uncompressed info
  // TBD: We could update the value from the information we already
  // have in stream_info[], but using the gpu results also updates
  // max_uncompressed_size to the actual uncompressed size, or zero if
  // decompression failed.
  compinfo.device_to_host(stream, true);

  const size_t num_columns = chunks.size().second;

  for (size_t i = 0; i < num_stripes; ++i) {
    for (size_t j = 0; j < num_columns; ++j) {
      auto& chunk = chunks[i][j];
      for (int k = 0; k < gpu::CI_NUM_STREAMS; ++k) {
        if (chunk.strm_len[k] > 0 && chunk.strm_id[k] < compinfo.size()) {
          chunk.streams[k]  = compinfo[chunk.strm_id[k]].uncompressed_data;
          chunk.strm_len[k] = compinfo[chunk.strm_id[k]].max_uncompressed_size;
        }
      }
    }
  }

  if (row_groups.size().first) {
    chunks.host_to_device(stream);
    row_groups.host_to_device(stream);
    gpu::ParseRowGroupIndex(row_groups.base_device_ptr(),
                            compinfo.device_ptr(),
                            chunks.base_device_ptr(),
                            num_columns,
                            num_stripes,
                            row_groups.size().first,
                            row_index_stride,
                            use_base_stride,
                            stream);
  }

  return decomp_data;
}

void reader::impl::decode_stream_data(cudf::detail::hostdevice_2dvector<gpu::ColumnDesc>& chunks,
                                      size_t num_dicts,
                                      size_t skip_rows,
                                      timezone_table_view tz_table,
                                      cudf::detail::hostdevice_2dvector<gpu::RowGroup>& row_groups,
                                      size_t row_index_stride,
                                      std::vector<column_buffer>& out_buffers,
                                      size_t level,
                                      rmm::cuda_stream_view stream)
{
  const auto num_stripes = chunks.size().first;
  const auto num_columns = chunks.size().second;

  // Update chunks with pointers to column data
  for (size_t i = 0; i < num_stripes; ++i) {
    for (size_t j = 0; j < num_columns; ++j) {
      auto& chunk            = chunks[i][j];
      chunk.column_data_base = out_buffers[j].data();
      chunk.valid_map_base   = out_buffers[j].null_mask();
    }
  }

  // Allocate global dictionary for deserializing
  rmm::device_uvector<gpu::DictionaryEntry> global_dict(num_dicts, stream);

  chunks.host_to_device(stream);
  gpu::DecodeNullsAndStringDictionaries(
    chunks.base_device_ptr(), global_dict.data(), num_columns, num_stripes, skip_rows, stream);
  gpu::DecodeOrcColumnData(chunks.base_device_ptr(),
                           global_dict.data(),
                           row_groups,
                           num_columns,
                           num_stripes,
                           skip_rows,
                           tz_table,
                           row_groups.size().first,
                           row_index_stride,
                           level,
                           stream);
  chunks.device_to_host(stream, true);

  for (size_t i = 0; i < num_stripes; ++i) {
    for (size_t j = 0; j < num_columns; ++j) {
      out_buffers[j].null_count() += chunks[i][j].null_count;
    }
  }
}

// Aggregate child column metadata per stripe and per column
void reader::impl::aggregate_child_meta(cudf::detail::host_2dspan<gpu::ColumnDesc> chunks,
                                        cudf::detail::host_2dspan<gpu::RowGroup> row_groups,
                                        std::vector<orc_column_meta> const& list_col,
                                        const int32_t level)
{
  const auto num_of_stripes         = chunks.size().first;
  const auto num_of_rowgroups       = row_groups.size().first;
  const auto num_parent_cols        = _selected_columns[level].size();
  const auto num_child_cols         = _selected_columns[level + 1].size();
  const auto number_of_child_chunks = num_child_cols * num_of_stripes;
  auto& num_child_rows              = _col_meta.num_child_rows;

  // Reset the meta to store child column details.
  num_child_rows.resize(_selected_columns[level + 1].size());
  std::fill(num_child_rows.begin(), num_child_rows.end(), 0);
  _col_meta.child_start_row.resize(number_of_child_chunks);
  _col_meta.num_child_rows_per_stripe.resize(number_of_child_chunks);
  _col_meta.rwgrp_meta.resize(num_of_rowgroups * num_child_cols);

  auto child_start_row = cudf::detail::host_2dspan<uint32_t>(
    _col_meta.child_start_row.data(), num_of_stripes, num_child_cols);
  auto num_child_rows_per_stripe = cudf::detail::host_2dspan<uint32_t>(
    _col_meta.num_child_rows_per_stripe.data(), num_of_stripes, num_child_cols);
  auto rwgrp_meta = cudf::detail::host_2dspan<reader_column_meta::row_group_meta>(
    _col_meta.rwgrp_meta.data(), num_of_rowgroups, num_child_cols);

  int index = 0;  // number of child column processed

  // For each parent column, update its child column meta for each stripe.
  std::for_each(list_col.cbegin(), list_col.cend(), [&](const auto p_col) {
    const auto parent_col_idx = _col_meta.orc_col_map[level][p_col.id];
    auto start_row            = 0;
    auto processed_row_groups = 0;

    for (size_t stripe_id = 0; stripe_id < num_of_stripes; stripe_id++) {
      // Aggregate num_rows and start_row from processed parent columns per row groups
      if (num_of_rowgroups) {
        auto stripe_num_row_groups = chunks[stripe_id][parent_col_idx].num_rowgroups;
        auto processed_child_rows  = 0;

        for (size_t rowgroup_id = 0; rowgroup_id < stripe_num_row_groups;
             rowgroup_id++, processed_row_groups++) {
          const auto child_rows = row_groups[processed_row_groups][parent_col_idx].num_child_rows;
          for (uint32_t id = 0; id < p_col.num_children; id++) {
            const auto child_col_idx                                  = index + id;
            rwgrp_meta[processed_row_groups][child_col_idx].start_row = processed_child_rows;
            rwgrp_meta[processed_row_groups][child_col_idx].num_rows  = child_rows;
          }
          processed_child_rows += child_rows;
        }
      }

      // Aggregate start row, number of rows per chunk and total number of rows in a column
      const auto child_rows = chunks[stripe_id][parent_col_idx].num_child_rows;
      for (uint32_t id = 0; id < p_col.num_children; id++) {
        const auto child_col_idx = index + id;

        num_child_rows[child_col_idx] += child_rows;
        num_child_rows_per_stripe[stripe_id][child_col_idx] = child_rows;
        // start row could be different for each column when there is nesting at each stripe level
        child_start_row[stripe_id][child_col_idx] = (stripe_id == 0) ? 0 : start_row;
      }
      start_row += child_rows;
    }
    index += p_col.num_children;
  });
}

std::unique_ptr<column> reader::impl::create_empty_column(const int32_t orc_col_id,
                                                          column_name_info& schema_info,
                                                          rmm::cuda_stream_view stream)
{
  schema_info.name = _metadata->get_column_name(0, orc_col_id);
  // If the column type is orc::DECIMAL see if the user
  // desires it to be converted to float64 or not
  auto const decimal_as_float64 = should_convert_decimal_column_to_float(
    _decimal_cols_as_float, _metadata->per_file_metadata[0], orc_col_id);
  auto const type = to_type_id(
    _metadata->get_schema(orc_col_id), _use_np_dtypes, _timestamp_type.id(), decimal_as_float64);
  int32_t scale = 0;
  std::vector<std::unique_ptr<column>> child_columns;
  std::unique_ptr<column> out_col = nullptr;

  switch (type) {
    case type_id::LIST:
      schema_info.children.emplace_back("offsets");
      schema_info.children.emplace_back("");
      out_col = make_lists_column(
        0,
        make_empty_column(data_type(type_id::INT32)),
        create_empty_column(
          _metadata->get_col_type(orc_col_id).subtypes[0], schema_info.children.back(), stream),
        0,
        rmm::device_buffer{0, stream},
        stream);

      break;

    case type_id::STRUCT:
      for (const auto col : _metadata->get_col_type(orc_col_id).subtypes) {
        schema_info.children.emplace_back("");
        child_columns.push_back(create_empty_column(col, schema_info.children.back(), stream));
      }
      out_col =
        make_structs_column(0, std::move(child_columns), 0, rmm::device_buffer{0, stream}, stream);
      break;

    case type_id::DECIMAL64:
      scale = -static_cast<int32_t>(_metadata->get_types()[orc_col_id].scale.value_or(0));
    default: out_col = make_empty_column(data_type(type, scale));
  }

  return out_col;
}

// Adds child column buffers to parent column
column_buffer&& reader::impl::assemble_buffer(const int32_t orc_col_id,
                                              std::vector<std::vector<column_buffer>>& col_buffers,
                                              const size_t level)
{
  auto const col_id = _col_meta.orc_col_map[level][orc_col_id];
  auto& col_buffer  = col_buffers[level][col_id];

  col_buffer.name = _metadata->get_column_name(0, orc_col_id);
  switch (col_buffer.type.id()) {
    case type_id::LIST:
      col_buffer.children.emplace_back(
        assemble_buffer(_metadata->get_col_type(orc_col_id).subtypes[0], col_buffers, level + 1));
      break;

    case type_id::STRUCT:
      for (auto const& col : _metadata->get_col_type(orc_col_id).subtypes) {
        col_buffer.children.emplace_back(assemble_buffer(col, col_buffers, level));
      }

      break;

    default: break;
  }

  return std::move(col_buffer);
}

// creates columns along with schema information for each column
void reader::impl::create_columns(std::vector<std::vector<column_buffer>>&& col_buffers,
                                  std::vector<std::unique_ptr<column>>& out_columns,
                                  std::vector<column_name_info>& schema_info,
                                  rmm::cuda_stream_view stream)
{
  for (size_t i = 0; i < _selected_columns[0].size();) {
    auto const& col_meta = _selected_columns[0][i];
    schema_info.emplace_back("");

    auto col_buffer = assemble_buffer(col_meta.id, col_buffers, 0);
    out_columns.emplace_back(make_column(col_buffer, &schema_info.back(), stream, _mr));

    // Need to skip child columns of struct which are at the same level and have been processed
    i += (col_buffers[0][i].type.id() == type_id::STRUCT) ? col_meta.num_children + 1 : 1;
  }
}

reader::impl::impl(std::vector<std::unique_ptr<datasource>>&& sources,
                   orc_reader_options const& options,
                   rmm::mr::device_memory_resource* mr)
  : _mr(mr), _sources(std::move(sources))
{
  // Open and parse the source(s) dataset metadata
  _metadata = std::make_unique<aggregate_orc_metadata>(_sources);

  // Select only columns required by the options
  _selected_columns =
    _metadata->select_columns(options.get_columns(), _has_timestamp_column, _has_list_column);

  // Override output timestamp resolution if requested
  if (options.get_timestamp_type().id() != type_id::EMPTY) {
    _timestamp_type = options.get_timestamp_type();
  }

  // Enable or disable attempt to use row index for parsing
  _use_index = options.is_enabled_use_index();

  // Enable or disable the conversion to numpy-compatible dtypes
  _use_np_dtypes = options.is_enabled_use_np_dtypes();

  // Control decimals conversion (float64 or int64 with optional scale)
  _decimal_cols_as_float = options.get_decimal_cols_as_float();
}

table_with_metadata reader::impl::read(size_type skip_rows,
                                       size_type num_rows,
                                       const std::vector<std::vector<size_type>>& stripes,
                                       rmm::cuda_stream_view stream)
{
  CUDF_EXPECTS(skip_rows == 0 or (not _has_list_column),
               "skip_rows is not supported by list column");

  std::vector<std::unique_ptr<column>> out_columns;
  // buffer and stripe data are stored as per nesting level
  std::vector<std::vector<column_buffer>> out_buffers(_selected_columns.size());
  std::vector<column_name_info> schema_info;
  std::vector<std::vector<rmm::device_buffer>> lvl_stripe_data(_selected_columns.size());
  table_metadata out_metadata;

  // There are no columns in the table
  if (_selected_columns.size() == 0) return {std::make_unique<table>(), std::move(out_metadata)};

  // Select only stripes required (aka row groups)
  const auto selected_stripes = _metadata->select_stripes(stripes, skip_rows, num_rows);

  // Iterates through levels of nested columns, struct columns and its children will be
  // in the same level since child column also have same number of rows,
  // list column children will be 1 level down compared to parent.
  for (size_t level = 0; level < _selected_columns.size(); level++) {
    auto& selected_columns = _selected_columns[level];
    // Association between each ORC column and its cudf::column
    _col_meta.orc_col_map.emplace_back(_metadata->get_num_cols(), -1);
    std::vector<orc_column_meta> list_col;

    // Get a list of column data types
    std::vector<data_type> column_types;
    for (auto& col : selected_columns) {
      // If the column type is orc::DECIMAL see if the user
      // desires it to be converted to float64 or not
      auto const decimal_as_float64 = should_convert_decimal_column_to_float(
        _decimal_cols_as_float, _metadata->per_file_metadata[0], col.id);
      auto col_type = to_type_id(
        _metadata->get_col_type(col.id), _use_np_dtypes, _timestamp_type.id(), decimal_as_float64);
      CUDF_EXPECTS(col_type != type_id::EMPTY, "Unknown type");
      // Remove this once we support Decimal128 data type
      CUDF_EXPECTS(
        (col_type != type_id::DECIMAL64) or (_metadata->get_col_type(col.id).precision <= 18),
        "Decimal data has precision > 18, Decimal64 data type doesn't support it.");
      if (col_type == type_id::DECIMAL64) {
        // sign of the scale is changed since cuDF follows c++ libraries like CNL
        // which uses negative scaling, but liborc and other libraries
        // follow positive scaling.
        auto const scale = -static_cast<int32_t>(_metadata->get_col_type(col.id).scale.value_or(0));
        column_types.emplace_back(col_type, scale);
      } else {
        column_types.emplace_back(col_type);
      }

      // Map each ORC column to its column
      _col_meta.orc_col_map[level][col.id] = column_types.size() - 1;
      if (col_type == type_id::LIST) list_col.emplace_back(col);
    }

    // If no rows or stripes to read, return empty columns
    if (num_rows <= 0 || selected_stripes.empty()) {
      for (size_t i = 0; i < _selected_columns[0].size();) {
        auto const& col_meta = _selected_columns[0][i];
        auto const schema    = _metadata->get_schema(col_meta.id);
        schema_info.emplace_back("");
        out_columns.push_back(
          std::move(create_empty_column(col_meta.id, schema_info.back(), stream)));
        // Since struct children will be in the same level, have to skip them.
        i += (schema.kind == orc::STRUCT) ? col_meta.num_children + 1 : 1;
      }
      break;
    } else {
      // Get the total number of stripes across all input files.
      size_t total_num_stripes =
        std::accumulate(selected_stripes.begin(),
                        selected_stripes.end(),
                        0,
                        [](size_t sum, auto& stripe_source_mapping) {
                          return sum + stripe_source_mapping.stripe_info.size();
                        });
      const auto num_columns = selected_columns.size();
      cudf::detail::hostdevice_2dvector<gpu::ColumnDesc> chunks(
        total_num_stripes, num_columns, stream);
      memset(chunks.base_host_ptr(), 0, chunks.memory_size());

      const bool use_index =
        (_use_index == true) &&
        // Only use if we don't have much work with complete columns & stripes
        // TODO: Consider nrows, gpu, and tune the threshold
        (num_rows > _metadata->get_row_index_stride() && !(_metadata->get_row_index_stride() & 7) &&
         _metadata->get_row_index_stride() > 0 && num_columns * total_num_stripes < 8 * 128) &&
        // Only use if first row is aligned to a stripe boundary
        // TODO: Fix logic to handle unaligned rows
        (skip_rows == 0);

      // Logically view streams as columns
      std::vector<orc_stream_info> stream_info;

      // Tracker for eventually deallocating compressed and uncompressed data
      auto& stripe_data = lvl_stripe_data[level];

      size_t stripe_start_row = 0;
      size_t num_dict_entries = 0;
      size_t num_rowgroups    = 0;
      int stripe_idx          = 0;

      for (auto const& stripe_source_mapping : selected_stripes) {
        // Iterate through the source files selected stripes
        for (auto const& stripe : stripe_source_mapping.stripe_info) {
          const auto stripe_info   = stripe.first;
          const auto stripe_footer = stripe.second;

          auto stream_count          = stream_info.size();
          const auto total_data_size = gather_stream_info(stripe_idx,
                                                          stripe_info,
                                                          stripe_footer,
                                                          _col_meta.orc_col_map[level],
                                                          selected_columns,
                                                          _metadata->get_types(),
                                                          use_index,
                                                          &num_dict_entries,
                                                          chunks,
                                                          stream_info);

          CUDF_EXPECTS(total_data_size > 0, "Expected streams data within stripe");

          stripe_data.emplace_back(total_data_size, stream);
          auto dst_base = static_cast<uint8_t*>(stripe_data.back().data());

          // Coalesce consecutive streams into one read
          while (stream_count < stream_info.size()) {
            const auto d_dst  = dst_base + stream_info[stream_count].dst_pos;
            const auto offset = stream_info[stream_count].offset;
            auto len          = stream_info[stream_count].length;
            stream_count++;

            while (stream_count < stream_info.size() &&
                   stream_info[stream_count].offset == offset + len) {
              len += stream_info[stream_count].length;
              stream_count++;
            }
            if (_metadata->per_file_metadata[stripe_source_mapping.source_idx]
                  .source->is_device_read_preferred(len)) {
              CUDF_EXPECTS(
                _metadata->per_file_metadata[stripe_source_mapping.source_idx].source->device_read(
                  offset, len, d_dst, stream) == len,
                "Unexpected discrepancy in bytes read.");
            } else {
              const auto buffer =
                _metadata->per_file_metadata[stripe_source_mapping.source_idx].source->host_read(
                  offset, len);
              CUDF_EXPECTS(buffer->size() == len, "Unexpected discrepancy in bytes read.");
              CUDA_TRY(hipMemcpyAsync(
                d_dst, buffer->data(), len, hipMemcpyHostToDevice, stream.value()));
              stream.synchronize();
            }
          }

          const auto num_rows_per_stripe = stripe_info->numberOfRows;
          const auto rowgroup_id         = num_rowgroups;
          auto stripe_num_rowgroups      = 0;
          if (use_index) {
            stripe_num_rowgroups = (num_rows_per_stripe + _metadata->get_row_index_stride() - 1) /
                                   _metadata->get_row_index_stride();
          }
          // Update chunks to reference streams pointers
          for (size_t col_idx = 0; col_idx < num_columns; col_idx++) {
            auto& chunk = chunks[stripe_idx][col_idx];
            // start row, number of rows in a each stripe and total number of rows
            // may change in lower levels of nesting
            chunk.start_row = (level == 0)
                                ? stripe_start_row
                                : _col_meta.child_start_row[stripe_idx * num_columns + col_idx];
            chunk.num_rows =
              (level == 0)
                ? stripe_info->numberOfRows
                : _col_meta.num_child_rows_per_stripe[stripe_idx * num_columns + col_idx];
            chunk.column_num_rows = (level == 0) ? num_rows : _col_meta.num_child_rows[col_idx];
            chunk.encoding_kind   = stripe_footer->columns[selected_columns[col_idx].id].kind;
            chunk.type_kind       = _metadata->per_file_metadata[stripe_source_mapping.source_idx]
                                .ff.types[selected_columns[col_idx].id]
                                .kind;
            auto const decimal_as_float64 =
              should_convert_decimal_column_to_float(_decimal_cols_as_float,
                                                     _metadata->per_file_metadata[0],
                                                     selected_columns[col_idx].id);
            chunk.decimal_scale = _metadata->per_file_metadata[stripe_source_mapping.source_idx]
                                    .ff.types[selected_columns[col_idx].id]
                                    .scale.value_or(0) |
                                  (decimal_as_float64 ? orc::gpu::orc_decimal2float64_scale : 0);

            chunk.rowgroup_id   = rowgroup_id;
            chunk.dtype_len     = (column_types[col_idx].id() == type_id::STRING)
                                    ? sizeof(string_index_pair)
                                  : ((column_types[col_idx].id() == type_id::LIST) or
                                 (column_types[col_idx].id() == type_id::STRUCT))
                                    ? sizeof(int32_t)
                                    : cudf::size_of(column_types[col_idx]);
            chunk.num_rowgroups = stripe_num_rowgroups;
            if (chunk.type_kind == orc::TIMESTAMP) {
              chunk.ts_clock_rate = to_clockrate(_timestamp_type.id());
            }
            for (int k = 0; k < gpu::CI_NUM_STREAMS; k++) {
              chunk.streams[k] = dst_base + stream_info[chunk.strm_id[k]].dst_pos;
            }
          }
          stripe_start_row += num_rows_per_stripe;
          num_rowgroups += stripe_num_rowgroups;

          stripe_idx++;
        }
      }

      // Process dataset chunk pages into output columns
      if (stripe_data.size() != 0) {
        auto row_groups =
          cudf::detail::hostdevice_2dvector<gpu::RowGroup>(num_rowgroups, num_columns, stream);
        if (level > 0 and row_groups.size().first) {
          cudf::host_span<gpu::RowGroup> row_groups_span(row_groups.base_host_ptr(),
                                                         num_rowgroups * num_columns);
          auto& rw_grp_meta = _col_meta.rwgrp_meta;

          // Update start row and num rows per row group
          std::transform(rw_grp_meta.begin(),
                         rw_grp_meta.end(),
                         row_groups_span.begin(),
                         rw_grp_meta.begin(),
                         [&](auto meta, auto& row_grp) {
                           row_grp.num_rows  = meta.num_rows;
                           row_grp.start_row = meta.start_row;
                           return meta;
                         });
        }
        // Setup row group descriptors if using indexes
        if (_metadata->per_file_metadata[0].ps.compression != orc::NONE) {
          auto decomp_data =
            decompress_stripe_data(chunks,
                                   stripe_data,
                                   _metadata->per_file_metadata[0].decompressor.get(),
                                   stream_info,
                                   total_num_stripes,
                                   row_groups,
                                   _metadata->get_row_index_stride(),
                                   level == 0,
                                   stream);
          stripe_data.clear();
          stripe_data.push_back(std::move(decomp_data));
        } else {
          if (row_groups.size().first) {
            chunks.host_to_device(stream);
            row_groups.host_to_device(stream);
            gpu::ParseRowGroupIndex(row_groups.base_device_ptr(),
                                    nullptr,
                                    chunks.base_device_ptr(),
                                    num_columns,
                                    total_num_stripes,
                                    num_rowgroups,
                                    _metadata->get_row_index_stride(),
                                    level == 0,
                                    stream);
          }
        }

        // Setup table for converting timestamp columns from local to UTC time
        auto const tz_table =
          _has_timestamp_column
            ? build_timezone_transition_table(
                selected_stripes[0].stripe_info[0].second->writerTimezone, stream)
            : timezone_table{};

        for (size_t i = 0; i < column_types.size(); ++i) {
          bool is_nullable = false;
          for (size_t j = 0; j < total_num_stripes; ++j) {
            if (chunks[j][i].strm_len[gpu::CI_PRESENT] != 0) {
              is_nullable = true;
              break;
            }
          }
          auto is_list_type = (column_types[i].id() == type_id::LIST);
          auto n_rows       = (level == 0) ? num_rows : _col_meta.num_child_rows[i];
          // For list column, offset column will be always size + 1
          if (is_list_type) n_rows++;
          out_buffers[level].emplace_back(column_types[i], n_rows, is_nullable, stream, _mr);
        }

        decode_stream_data(chunks,
                           num_dict_entries,
                           skip_rows,
                           tz_table.view(),
                           row_groups,
                           _metadata->get_row_index_stride(),
                           out_buffers[level],
                           level,
                           stream);

        // Extract information to process list child columns
        if (list_col.size()) {
          row_groups.device_to_host(stream, true);
          aggregate_child_meta(chunks, row_groups, list_col, level);
        }

        // ORC stores number of elements at each row, so we need to generate offsets from that
        if (list_col.size()) {
          std::vector<list_buffer_data> buff_data;
          std::for_each(
            out_buffers[level].begin(), out_buffers[level].end(), [&buff_data](auto& out_buffer) {
              if (out_buffer.type.id() == type_id::LIST) {
                auto data = static_cast<size_type*>(out_buffer.data());
                buff_data.emplace_back(list_buffer_data{data, out_buffer.size});
              }
            });

          auto const dev_buff_data = cudf::detail::make_device_uvector_async(buff_data, stream);
          generate_offsets_for_list(dev_buff_data, stream);
        }
      }
    }
  }

  // If out_columns is empty, then create columns from buffer.
  if (out_columns.empty()) {
    create_columns(std::move(out_buffers), out_columns, schema_info, stream);
  }

  // Return column names (must match order of returned columns)
  out_metadata.column_names.reserve(schema_info.size());
  std::transform(schema_info.cbegin(),
                 schema_info.cend(),
                 std::back_inserter(out_metadata.column_names),
                 [](auto info) { return info.name; });

  out_metadata.schema_info = std::move(schema_info);

  for (const auto& meta : _metadata->per_file_metadata) {
    for (const auto& kv : meta.ff.metadata) {
      out_metadata.user_data.insert({kv.name, kv.value});
    }
  }

  return {std::make_unique<table>(std::move(out_columns)), std::move(out_metadata)};
}

// Forward to implementation
reader::reader(std::vector<std::string> const& filepaths,
               orc_reader_options const& options,
               rmm::cuda_stream_view stream,
               rmm::mr::device_memory_resource* mr)
{
  _impl = std::make_unique<impl>(datasource::create(filepaths), options, mr);
}

// Forward to implementation
reader::reader(std::vector<std::unique_ptr<cudf::io::datasource>>&& sources,
               orc_reader_options const& options,
               rmm::cuda_stream_view stream,
               rmm::mr::device_memory_resource* mr)
{
  _impl = std::make_unique<impl>(std::move(sources), options, mr);
}

// Destructor within this translation unit
reader::~reader() = default;

// Forward to implementation
table_with_metadata reader::read(orc_reader_options const& options, rmm::cuda_stream_view stream)
{
  return _impl->read(
    options.get_skip_rows(), options.get_num_rows(), options.get_stripes(), stream);
}
}  // namespace orc
}  // namespace detail
}  // namespace io
}  // namespace cudf
