#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019-2022, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

/**
 * @file reader_impl.cu
 * @brief cuDF-IO ORC reader class implementation
 */

#include "orc.hpp"
#include "orc_gpu.hpp"

#include "reader_impl.hpp"
#include "timezone.cuh"

#include <io/comp/gpuinflate.hpp>
#include <io/comp/nvcomp_adapter.hpp>
#include <io/utilities/config_utils.hpp>
#include <io/utilities/time_utils.cuh>

#include <cudf/detail/utilities/integer_utils.hpp>
#include <cudf/detail/utilities/vector_factories.hpp>
#include <cudf/table/table.hpp>
#include <cudf/utilities/bit.hpp>
#include <cudf/utilities/error.hpp>
#include <cudf/utilities/traits.hpp>
#include <rmm/device_scalar.hpp>

#include <rmm/cuda_stream_view.hpp>
#include <rmm/device_buffer.hpp>
#include <rmm/device_uvector.hpp>
#include <rmm/exec_policy.hpp>

#include <thrust/copy.h>
#include <thrust/execution_policy.h>
#include <thrust/for_each.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/iterator/zip_iterator.h>
#include <thrust/pair.h>
#include <thrust/scan.h>
#include <thrust/transform.h>
#include <thrust/tuple.h>

#include <algorithm>
#include <iterator>

namespace cudf {
namespace io {
namespace detail {
namespace orc {
using namespace cudf::io::orc;

namespace {
/**
 * @brief Function that translates ORC data kind to cuDF type enum
 */
constexpr type_id to_type_id(const orc::SchemaType& schema,
                             bool use_np_dtypes,
                             type_id timestamp_type_id,
                             type_id decimal_type_id)
{
  switch (schema.kind) {
    case orc::BOOLEAN: return type_id::BOOL8;
    case orc::BYTE: return type_id::INT8;
    case orc::SHORT: return type_id::INT16;
    case orc::INT: return type_id::INT32;
    case orc::LONG: return type_id::INT64;
    case orc::FLOAT: return type_id::FLOAT32;
    case orc::DOUBLE: return type_id::FLOAT64;
    case orc::STRING:
    case orc::BINARY:
    case orc::VARCHAR:
    case orc::CHAR:
      // Variable-length types can all be mapped to STRING
      return type_id::STRING;
    case orc::TIMESTAMP:
      return (timestamp_type_id != type_id::EMPTY) ? timestamp_type_id
                                                   : type_id::TIMESTAMP_NANOSECONDS;
    case orc::DATE:
      // There isn't a (DAYS -> np.dtype) mapping
      return (use_np_dtypes) ? type_id::TIMESTAMP_MILLISECONDS : type_id::TIMESTAMP_DAYS;
    case orc::DECIMAL: return decimal_type_id;
    // Need to update once cuDF plans to support map type
    case orc::MAP:
    case orc::LIST: return type_id::LIST;
    case orc::STRUCT: return type_id::STRUCT;
    default: break;
  }

  return type_id::EMPTY;
}

constexpr std::pair<gpu::StreamIndexType, uint32_t> get_index_type_and_pos(
  const orc::StreamKind kind, uint32_t skip_count, bool non_child)
{
  switch (kind) {
    case orc::DATA:
      skip_count += 1;
      skip_count |= (skip_count & 0xff) << 8;
      return std::pair(gpu::CI_DATA, skip_count);
    case orc::LENGTH:
    case orc::SECONDARY:
      skip_count += 1;
      skip_count |= (skip_count & 0xff) << 16;
      return std::pair(gpu::CI_DATA2, skip_count);
    case orc::DICTIONARY_DATA: return std::pair(gpu::CI_DICTIONARY, skip_count);
    case orc::PRESENT:
      skip_count += (non_child ? 1 : 0);
      return std::pair(gpu::CI_PRESENT, skip_count);
    case orc::ROW_INDEX: return std::pair(gpu::CI_INDEX, skip_count);
    default:
      // Skip this stream as it's not strictly required
      return std::pair(gpu::CI_NUM_STREAMS, 0);
  }
}

/**
 * @brief struct to store buffer data and size of list buffer
 */
struct list_buffer_data {
  size_type* data;
  size_type size;
};

// Generates offsets for list buffer from number of elements in a row.
void generate_offsets_for_list(rmm::device_uvector<list_buffer_data> const& buff_data,
                               rmm::cuda_stream_view stream)
{
  auto transformer = [] __device__(list_buffer_data list_data) {
    thrust::exclusive_scan(
      thrust::seq, list_data.data, list_data.data + list_data.size, list_data.data);
  };
  thrust::for_each(rmm::exec_policy(stream), buff_data.begin(), buff_data.end(), transformer);
  stream.synchronize();
}

/**
 * @brief Struct that maps ORC streams to columns
 */
struct orc_stream_info {
  orc_stream_info() = default;
  explicit orc_stream_info(
    uint64_t offset_, size_t dst_pos_, uint32_t length_, uint32_t gdf_idx_, uint32_t stripe_idx_)
    : offset(offset_),
      dst_pos(dst_pos_),
      length(length_),
      gdf_idx(gdf_idx_),
      stripe_idx(stripe_idx_)
  {
  }
  uint64_t offset;      // offset in file
  size_t dst_pos;       // offset in memory relative to start of compressed stripe data
  size_t length;        // length in file
  uint32_t gdf_idx;     // column index
  uint32_t stripe_idx;  // stripe index
};

/**
 * @brief Function that populates column descriptors stream/chunk
 */
size_t gather_stream_info(const size_t stripe_index,
                          const orc::StripeInformation* stripeinfo,
                          const orc::StripeFooter* stripefooter,
                          const std::vector<int>& orc2gdf,
                          const std::vector<orc::SchemaType> types,
                          bool use_index,
                          size_t* num_dictionary_entries,
                          cudf::detail::hostdevice_2dvector<gpu::ColumnDesc>& chunks,
                          std::vector<orc_stream_info>& stream_info,
                          bool apply_struct_map)
{
  uint64_t src_offset = 0;
  uint64_t dst_offset = 0;
  for (const auto& stream : stripefooter->streams) {
    if (!stream.column_id || *stream.column_id >= orc2gdf.size()) {
      dst_offset += stream.length;
      continue;
    }

    auto const column_id = *stream.column_id;
    auto col             = orc2gdf[column_id];

    if (col == -1 and apply_struct_map) {
      // A struct-type column has no data itself, but rather child columns
      // for each of its fields. There is only a PRESENT stream, which
      // needs to be included for the reader.
      const auto schema_type = types[column_id];
      if (schema_type.subtypes.size() != 0) {
        if (schema_type.kind == orc::STRUCT && stream.kind == orc::PRESENT) {
          for (const auto& idx : schema_type.subtypes) {
            auto child_idx = (idx < orc2gdf.size()) ? orc2gdf[idx] : -1;
            if (child_idx >= 0) {
              col                             = child_idx;
              auto& chunk                     = chunks[stripe_index][col];
              chunk.strm_id[gpu::CI_PRESENT]  = stream_info.size();
              chunk.strm_len[gpu::CI_PRESENT] = stream.length;
            }
          }
        }
      }
    }
    if (col != -1) {
      if (src_offset >= stripeinfo->indexLength || use_index) {
        // NOTE: skip_count field is temporarily used to track index ordering
        auto& chunk = chunks[stripe_index][col];
        const auto idx =
          get_index_type_and_pos(stream.kind, chunk.skip_count, col == orc2gdf[column_id]);
        if (idx.first < gpu::CI_NUM_STREAMS) {
          chunk.strm_id[idx.first]  = stream_info.size();
          chunk.strm_len[idx.first] = stream.length;
          chunk.skip_count          = idx.second;

          if (idx.first == gpu::CI_DICTIONARY) {
            chunk.dictionary_start = *num_dictionary_entries;
            chunk.dict_len         = stripefooter->columns[column_id].dictionarySize;
            *num_dictionary_entries += stripefooter->columns[column_id].dictionarySize;
          }
        }
      }
      stream_info.emplace_back(
        stripeinfo->offset + src_offset, dst_offset, stream.length, col, stripe_index);
      dst_offset += stream.length;
    }
    src_offset += stream.length;
  }

  return dst_offset;
}

/**
 * @brief Determines cuDF type of an ORC Decimal column.
 */
auto decimal_column_type(std::vector<std::string> const& decimal128_columns,
                         cudf::io::orc::detail::aggregate_orc_metadata const& metadata,
                         int column_index)
{
  if (metadata.get_col_type(column_index).kind != DECIMAL) { return type_id::EMPTY; }

  if (std::find(decimal128_columns.cbegin(),
                decimal128_columns.cend(),
                metadata.column_path(0, column_index)) != decimal128_columns.end()) {
    return type_id::DECIMAL128;
  }

  auto const precision = metadata.get_col_type(column_index)
                           .precision.value_or(cuda::std::numeric_limits<int64_t>::digits10);
  if (precision <= cuda::std::numeric_limits<int32_t>::digits10) { return type_id::DECIMAL32; }
  if (precision <= cuda::std::numeric_limits<int64_t>::digits10) { return type_id::DECIMAL64; }
  return type_id::DECIMAL128;
}

}  // namespace

__global__ void decompress_check_kernel(device_span<compression_result const> results,
                                        bool* any_block_failure)
{
  auto tid = blockIdx.x * blockDim.x + threadIdx.x;
  if (tid < results.size()) {
    if (results[tid].status != compression_status::SUCCESS) {
      *any_block_failure = true;  // Doesn't need to be atomic
    }
  }
}

void decompress_check(device_span<compression_result> results,
                      bool* any_block_failure,
                      rmm::cuda_stream_view stream)
{
  if (results.empty()) { return; }  // early exit for empty results

  dim3 block(128);
  dim3 grid(cudf::util::div_rounding_up_safe(results.size(), static_cast<size_t>(block.x)));
  decompress_check_kernel<<<grid, block, 0, stream.value()>>>(results, any_block_failure);
}

rmm::device_buffer reader::impl::decompress_stripe_data(
  cudf::detail::hostdevice_2dvector<gpu::ColumnDesc>& chunks,
  const std::vector<rmm::device_buffer>& stripe_data,
  OrcDecompressor const& decompressor,
  std::vector<orc_stream_info>& stream_info,
  size_t num_stripes,
  cudf::detail::hostdevice_2dvector<gpu::RowGroup>& row_groups,
  size_t row_index_stride,
  bool use_base_stride,
  rmm::cuda_stream_view stream)
{
  // For checking whether we decompress successfully
  hostdevice_vector<bool> any_block_failure(1, stream);
  any_block_failure[0] = false;
  any_block_failure.host_to_device(stream);

  // Parse the columns' compressed info
  hostdevice_vector<gpu::CompressedStreamInfo> compinfo(0, stream_info.size(), stream);
  for (const auto& info : stream_info) {
    compinfo.push_back(gpu::CompressedStreamInfo(
      static_cast<const uint8_t*>(stripe_data[info.stripe_idx].data()) + info.dst_pos,
      info.length));
  }
  compinfo.host_to_device(stream);

  // Workaround for ZSTD. It is possible to have compression ratios > 2048:1,
  // so the heuristic in gpuParseCompressedStripeData() to estimate the size for
  // small blocks can be too low. Disable the estimation for ZSTD.
  auto allow_block_size_estimate = (decompressor.compression() != compression_type::ZSTD);
  gpu::ParseCompressedStripeData(compinfo.device_ptr(),
                                 compinfo.size(),
                                 decompressor.GetBlockSize(),
                                 decompressor.GetLog2MaxCompressionRatio(),
                                 allow_block_size_estimate,
                                 stream);
  compinfo.device_to_host(stream, true);

  // Count the exact number of compressed blocks
  size_t num_compressed_blocks   = 0;
  size_t num_uncompressed_blocks = 0;
  size_t total_decomp_size       = 0;
  for (size_t i = 0; i < compinfo.size(); ++i) {
    num_compressed_blocks += compinfo[i].num_compressed_blocks;
    num_uncompressed_blocks += compinfo[i].num_uncompressed_blocks;
    total_decomp_size += compinfo[i].max_uncompressed_size;
  }
  CUDF_EXPECTS(total_decomp_size > 0, "No decompressible data found");

  rmm::device_buffer decomp_data(total_decomp_size, stream);
  rmm::device_uvector<device_span<uint8_t const>> inflate_in(
    num_compressed_blocks + num_uncompressed_blocks, stream);
  rmm::device_uvector<device_span<uint8_t>> inflate_out(
    num_compressed_blocks + num_uncompressed_blocks, stream);
  rmm::device_uvector<compression_result> inflate_res(num_compressed_blocks, stream);
  thrust::fill(rmm::exec_policy(stream),
               inflate_res.begin(),
               inflate_res.end(),
               compression_result{0, compression_status::FAILURE});

  // Parse again to populate the decompression input/output buffers
  size_t decomp_offset           = 0;
  uint32_t max_uncomp_block_size = 0;
  uint32_t start_pos             = 0;
  auto start_pos_uncomp          = (uint32_t)num_compressed_blocks;
  for (size_t i = 0; i < compinfo.size(); ++i) {
    auto dst_base                 = static_cast<uint8_t*>(decomp_data.data());
    compinfo[i].uncompressed_data = dst_base + decomp_offset;
    compinfo[i].dec_in_ctl        = inflate_in.data() + start_pos;
    compinfo[i].dec_out_ctl       = inflate_out.data() + start_pos;
    compinfo[i].dec_res      = {inflate_res.data() + start_pos, compinfo[i].num_compressed_blocks};
    compinfo[i].copy_in_ctl  = inflate_in.data() + start_pos_uncomp;
    compinfo[i].copy_out_ctl = inflate_out.data() + start_pos_uncomp;

    stream_info[i].dst_pos = decomp_offset;
    decomp_offset += compinfo[i].max_uncompressed_size;
    start_pos += compinfo[i].num_compressed_blocks;
    start_pos_uncomp += compinfo[i].num_uncompressed_blocks;
    max_uncomp_block_size =
      std::max(max_uncomp_block_size, compinfo[i].max_uncompressed_block_size);
  }
  compinfo.host_to_device(stream);
  gpu::ParseCompressedStripeData(compinfo.device_ptr(),
                                 compinfo.size(),
                                 decompressor.GetBlockSize(),
                                 decompressor.GetLog2MaxCompressionRatio(),
                                 allow_block_size_estimate,
                                 stream);

  // Dispatch batches of blocks to decompress
  if (num_compressed_blocks > 0) {
    device_span<device_span<uint8_t const>> inflate_in_view{inflate_in.data(),
                                                            num_compressed_blocks};
    device_span<device_span<uint8_t>> inflate_out_view{inflate_out.data(), num_compressed_blocks};
    switch (decompressor.compression()) {
      case compression_type::ZLIB:
        if (nvcomp::is_decompression_disabled(nvcomp::compression_type::DEFLATE)) {
          gpuinflate(
            inflate_in_view, inflate_out_view, inflate_res, gzip_header_included::NO, stream);
        } else {
          nvcomp::batched_decompress(nvcomp::compression_type::DEFLATE,
                                     inflate_in_view,
                                     inflate_out_view,
                                     inflate_res,
                                     max_uncomp_block_size,
                                     total_decomp_size,
                                     stream);
        }
        break;
      case compression_type::SNAPPY:
        if (nvcomp::is_decompression_disabled(nvcomp::compression_type::SNAPPY)) {
          gpu_unsnap(inflate_in_view, inflate_out_view, inflate_res, stream);
        } else {
          nvcomp::batched_decompress(nvcomp::compression_type::SNAPPY,
                                     inflate_in_view,
                                     inflate_out_view,
                                     inflate_res,
                                     max_uncomp_block_size,
                                     total_decomp_size,
                                     stream);
        }
        break;
      case compression_type::ZSTD:
        if (auto const reason = nvcomp::is_decompression_disabled(nvcomp::compression_type::ZSTD);
            reason) {
          CUDF_FAIL("Decompression error: " + reason.value());
        }
        nvcomp::batched_decompress(nvcomp::compression_type::ZSTD,
                                   inflate_in_view,
                                   inflate_out_view,
                                   inflate_res,
                                   max_uncomp_block_size,
                                   total_decomp_size,
                                   stream);
        break;
      default: CUDF_FAIL("Unexpected decompression dispatch"); break;
    }
    decompress_check(inflate_res, any_block_failure.device_ptr(), stream);
  }
  if (num_uncompressed_blocks > 0) {
    device_span<device_span<uint8_t const>> copy_in_view{inflate_in.data() + num_compressed_blocks,
                                                         num_uncompressed_blocks};
    device_span<device_span<uint8_t>> copy_out_view{inflate_out.data() + num_compressed_blocks,
                                                    num_uncompressed_blocks};
    gpu_copy_uncompressed_blocks(copy_in_view, copy_out_view, stream);
  }
  gpu::PostDecompressionReassemble(compinfo.device_ptr(), compinfo.size(), stream);

  any_block_failure.device_to_host(stream);

  compinfo.device_to_host(stream, true);

  // We can check on host after stream synchronize
  CUDF_EXPECTS(not any_block_failure[0], "Error during decompression");

  const size_t num_columns = chunks.size().second;

  // Update the stream information with the updated uncompressed info
  // TBD: We could update the value from the information we already
  // have in stream_info[], but using the gpu results also updates
  // max_uncompressed_size to the actual uncompressed size, or zero if
  // decompression failed.
  for (size_t i = 0; i < num_stripes; ++i) {
    for (size_t j = 0; j < num_columns; ++j) {
      auto& chunk = chunks[i][j];
      for (int k = 0; k < gpu::CI_NUM_STREAMS; ++k) {
        if (chunk.strm_len[k] > 0 && chunk.strm_id[k] < compinfo.size()) {
          chunk.streams[k]  = compinfo[chunk.strm_id[k]].uncompressed_data;
          chunk.strm_len[k] = compinfo[chunk.strm_id[k]].max_uncompressed_size;
        }
      }
    }
  }

  if (row_groups.size().first) {
    chunks.host_to_device(stream);
    row_groups.host_to_device(stream);
    gpu::ParseRowGroupIndex(row_groups.base_device_ptr(),
                            compinfo.device_ptr(),
                            chunks.base_device_ptr(),
                            num_columns,
                            num_stripes,
                            row_groups.size().first,
                            row_index_stride,
                            use_base_stride,
                            stream);
  }

  return decomp_data;
}

/**
 * @brief Updates null mask of columns whose parent is a struct column.
 *        If struct column has null element, that row would be
 *        skipped while writing child column in ORC, so we need to insert the missing null
 *        elements in child column.
 *        There is another behavior from pyspark, where if the child column doesn't have any null
 *        elements, it will not have present stream, so in that case parent null mask need to be
 *        copied to child column.
 *
 * @param chunks Vector of list of column chunk descriptors
 * @param out_buffers Output columns' device buffers
 * @param stream CUDA stream used for device memory operations and kernel launches.
 * @param mr Device memory resource to use for device memory allocation
 */
void update_null_mask(cudf::detail::hostdevice_2dvector<gpu::ColumnDesc>& chunks,
                      std::vector<column_buffer>& out_buffers,
                      rmm::cuda_stream_view stream,
                      rmm::mr::device_memory_resource* mr)
{
  const auto num_stripes = chunks.size().first;
  const auto num_columns = chunks.size().second;
  bool is_mask_updated   = false;

  for (size_t col_idx = 0; col_idx < num_columns; ++col_idx) {
    if (chunks[0][col_idx].parent_validity_info.valid_map_base != nullptr) {
      if (not is_mask_updated) {
        chunks.device_to_host(stream, true);
        is_mask_updated = true;
      }

      auto parent_valid_map_base = chunks[0][col_idx].parent_validity_info.valid_map_base;
      auto child_valid_map_base  = out_buffers[col_idx].null_mask();
      auto child_mask_len =
        chunks[0][col_idx].column_num_rows - chunks[0][col_idx].parent_validity_info.null_count;
      auto parent_mask_len = chunks[0][col_idx].column_num_rows;

      if (child_valid_map_base != nullptr) {
        rmm::device_uvector<uint32_t> dst_idx(child_mask_len, stream);
        // Copy indexes at which the parent has valid value.
        thrust::copy_if(rmm::exec_policy(stream),
                        thrust::make_counting_iterator(0),
                        thrust::make_counting_iterator(0) + parent_mask_len,
                        dst_idx.begin(),
                        [parent_valid_map_base] __device__(auto idx) {
                          return bit_is_set(parent_valid_map_base, idx);
                        });

        auto merged_null_mask = cudf::detail::create_null_mask(
          parent_mask_len, mask_state::ALL_NULL, rmm::cuda_stream_view(stream), mr);
        auto merged_mask      = static_cast<bitmask_type*>(merged_null_mask.data());
        uint32_t* dst_idx_ptr = dst_idx.data();
        // Copy child valid bits from child column to valid indexes, this will merge both child
        // and parent null masks
        thrust::for_each(rmm::exec_policy(stream),
                         thrust::make_counting_iterator(0),
                         thrust::make_counting_iterator(0) + dst_idx.size(),
                         [child_valid_map_base, dst_idx_ptr, merged_mask] __device__(auto idx) {
                           if (bit_is_set(child_valid_map_base, idx)) {
                             cudf::set_bit(merged_mask, dst_idx_ptr[idx]);
                           };
                         });

        out_buffers[col_idx]._null_mask = std::move(merged_null_mask);

      } else {
        // Since child column doesn't have a mask, copy parent null mask
        auto mask_size = bitmask_allocation_size_bytes(parent_mask_len);
        out_buffers[col_idx]._null_mask =
          rmm::device_buffer(static_cast<void*>(parent_valid_map_base), mask_size, stream, mr);
      }
    }
  }

  thrust::counting_iterator<int> col_idx_it(0);
  thrust::counting_iterator<int> stripe_idx_it(0);

  if (is_mask_updated) {
    // Update chunks with pointers to column data which might have been changed.
    std::for_each(stripe_idx_it, stripe_idx_it + num_stripes, [&](auto stripe_idx) {
      std::for_each(col_idx_it, col_idx_it + num_columns, [&](auto col_idx) {
        auto& chunk          = chunks[stripe_idx][col_idx];
        chunk.valid_map_base = out_buffers[col_idx].null_mask();
      });
    });
    chunks.host_to_device(stream, true);
  }
}

/**
 * @brief Compute the per-stripe prefix sum of null count, for each struct column in the current
 * layer.
 */
void scan_null_counts(cudf::detail::hostdevice_2dvector<gpu::ColumnDesc> const& chunks,
                      cudf::host_span<rmm::device_uvector<uint32_t>> prefix_sums,
                      rmm::cuda_stream_view stream)
{
  auto const num_stripes = chunks.size().first;
  if (num_stripes == 0) return;

  auto const num_columns = chunks.size().second;
  std::vector<thrust::pair<size_type, cudf::device_span<uint32_t>>> prefix_sums_to_update;
  for (auto col_idx = 0ul; col_idx < num_columns; ++col_idx) {
    // Null counts sums are only needed for children of struct columns
    if (chunks[0][col_idx].type_kind == STRUCT) {
      prefix_sums_to_update.emplace_back(col_idx, prefix_sums[col_idx]);
    }
  }
  auto const d_prefix_sums_to_update =
    cudf::detail::make_device_uvector_async(prefix_sums_to_update, stream);

  thrust::for_each(rmm::exec_policy(stream),
                   d_prefix_sums_to_update.begin(),
                   d_prefix_sums_to_update.end(),
                   [chunks = cudf::detail::device_2dspan<gpu::ColumnDesc const>{chunks}] __device__(
                     auto const& idx_psums) {
                     auto const col_idx = idx_psums.first;
                     auto const psums   = idx_psums.second;

                     thrust::transform(
                       thrust::seq,
                       thrust::make_counting_iterator(0),
                       thrust::make_counting_iterator(0) + psums.size(),
                       psums.begin(),
                       [&](auto stripe_idx) { return chunks[stripe_idx][col_idx].null_count; });

                     thrust::inclusive_scan(thrust::seq, psums.begin(), psums.end(), psums.begin());
                   });
  // `prefix_sums_to_update` goes out of scope, copy has to be done before we return
  stream.synchronize();
}

void reader::impl::decode_stream_data(cudf::detail::hostdevice_2dvector<gpu::ColumnDesc>& chunks,
                                      size_t num_dicts,
                                      size_t skip_rows,
                                      timezone_table_view tz_table,
                                      cudf::detail::hostdevice_2dvector<gpu::RowGroup>& row_groups,
                                      size_t row_index_stride,
                                      std::vector<column_buffer>& out_buffers,
                                      size_t level,
                                      rmm::cuda_stream_view stream)
{
  const auto num_stripes = chunks.size().first;
  const auto num_columns = chunks.size().second;
  thrust::counting_iterator<int> col_idx_it(0);
  thrust::counting_iterator<int> stripe_idx_it(0);

  // Update chunks with pointers to column data
  std::for_each(stripe_idx_it, stripe_idx_it + num_stripes, [&](auto stripe_idx) {
    std::for_each(col_idx_it, col_idx_it + num_columns, [&](auto col_idx) {
      auto& chunk            = chunks[stripe_idx][col_idx];
      chunk.column_data_base = out_buffers[col_idx].data();
      chunk.valid_map_base   = out_buffers[col_idx].null_mask();
    });
  });

  // Allocate global dictionary for deserializing
  rmm::device_uvector<gpu::DictionaryEntry> global_dict(num_dicts, stream);

  chunks.host_to_device(stream, true);
  gpu::DecodeNullsAndStringDictionaries(
    chunks.base_device_ptr(), global_dict.data(), num_columns, num_stripes, skip_rows, stream);

  if (level > 0) {
    // Update nullmasks for children if parent was a struct and had null mask
    update_null_mask(chunks, out_buffers, stream, _mr);
  }

  rmm::device_scalar<size_type> error_count(0, stream);
  // Update the null map for child columns
  gpu::DecodeOrcColumnData(chunks.base_device_ptr(),
                           global_dict.data(),
                           row_groups,
                           num_columns,
                           num_stripes,
                           skip_rows,
                           tz_table,
                           row_groups.size().first,
                           row_index_stride,
                           level,
                           error_count.data(),
                           stream);
  chunks.device_to_host(stream);
  // `value` synchronizes
  auto const num_errors = error_count.value(stream);
  CUDF_EXPECTS(num_errors == 0, "ORC data decode failed");

  std::for_each(col_idx_it + 0, col_idx_it + num_columns, [&](auto col_idx) {
    out_buffers[col_idx].null_count() =
      std::accumulate(stripe_idx_it + 0,
                      stripe_idx_it + num_stripes,
                      0,
                      [&](auto null_count, auto const stripe_idx) {
                        return null_count + chunks[stripe_idx][col_idx].null_count;
                      });
  });
}

// Aggregate child column metadata per stripe and per column
void reader::impl::aggregate_child_meta(cudf::detail::host_2dspan<gpu::ColumnDesc> chunks,
                                        cudf::detail::host_2dspan<gpu::RowGroup> row_groups,
                                        std::vector<column_buffer>& out_buffers,
                                        std::vector<orc_column_meta> const& list_col,
                                        const size_type level)
{
  const auto num_of_stripes         = chunks.size().first;
  const auto num_of_rowgroups       = row_groups.size().first;
  const auto num_parent_cols        = selected_columns.levels[level].size();
  const auto num_child_cols         = selected_columns.levels[level + 1].size();
  const auto number_of_child_chunks = num_child_cols * num_of_stripes;
  auto& num_child_rows              = _col_meta.num_child_rows;
  auto& parent_column_data          = _col_meta.parent_column_data;

  // Reset the meta to store child column details.
  num_child_rows.resize(selected_columns.levels[level + 1].size());
  std::fill(num_child_rows.begin(), num_child_rows.end(), 0);
  parent_column_data.resize(number_of_child_chunks);
  _col_meta.parent_column_index.resize(number_of_child_chunks);
  _col_meta.child_start_row.resize(number_of_child_chunks);
  _col_meta.num_child_rows_per_stripe.resize(number_of_child_chunks);
  _col_meta.rwgrp_meta.resize(num_of_rowgroups * num_child_cols);

  auto child_start_row = cudf::detail::host_2dspan<uint32_t>(
    _col_meta.child_start_row.data(), num_of_stripes, num_child_cols);
  auto num_child_rows_per_stripe = cudf::detail::host_2dspan<uint32_t>(
    _col_meta.num_child_rows_per_stripe.data(), num_of_stripes, num_child_cols);
  auto rwgrp_meta = cudf::detail::host_2dspan<reader_column_meta::row_group_meta>(
    _col_meta.rwgrp_meta.data(), num_of_rowgroups, num_child_cols);

  int index = 0;  // number of child column processed

  // For each parent column, update its child column meta for each stripe.
  std::for_each(list_col.cbegin(), list_col.cend(), [&](const auto p_col) {
    const auto parent_col_idx = _col_meta.orc_col_map[level][p_col.id];
    auto start_row            = 0;
    auto processed_row_groups = 0;

    for (size_t stripe_id = 0; stripe_id < num_of_stripes; stripe_id++) {
      // Aggregate num_rows and start_row from processed parent columns per row groups
      if (num_of_rowgroups) {
        auto stripe_num_row_groups = chunks[stripe_id][parent_col_idx].num_rowgroups;
        auto processed_child_rows  = 0;

        for (size_t rowgroup_id = 0; rowgroup_id < stripe_num_row_groups;
             rowgroup_id++, processed_row_groups++) {
          const auto child_rows = row_groups[processed_row_groups][parent_col_idx].num_child_rows;
          for (size_type id = 0; id < p_col.num_children; id++) {
            const auto child_col_idx                                  = index + id;
            rwgrp_meta[processed_row_groups][child_col_idx].start_row = processed_child_rows;
            rwgrp_meta[processed_row_groups][child_col_idx].num_rows  = child_rows;
          }
          processed_child_rows += child_rows;
        }
      }

      // Aggregate start row, number of rows per chunk and total number of rows in a column
      const auto child_rows = chunks[stripe_id][parent_col_idx].num_child_rows;
      for (size_type id = 0; id < p_col.num_children; id++) {
        const auto child_col_idx = index + id;

        num_child_rows[child_col_idx] += child_rows;
        num_child_rows_per_stripe[stripe_id][child_col_idx] = child_rows;
        // start row could be different for each column when there is nesting at each stripe level
        child_start_row[stripe_id][child_col_idx] = (stripe_id == 0) ? 0 : start_row;
      }
      start_row += child_rows;
    }

    // Parent column null mask and null count would be required for child column
    // to adjust its nullmask.
    auto type              = out_buffers[parent_col_idx].type.id();
    auto parent_null_count = static_cast<uint32_t>(out_buffers[parent_col_idx].null_count());
    auto parent_valid_map  = out_buffers[parent_col_idx].null_mask();
    auto num_rows          = out_buffers[parent_col_idx].size;

    for (size_type id = 0; id < p_col.num_children; id++) {
      const auto child_col_idx                     = index + id;
      _col_meta.parent_column_index[child_col_idx] = parent_col_idx;
      if (type == type_id::STRUCT) {
        parent_column_data[child_col_idx] = {parent_valid_map, parent_null_count};
        // Number of rows in child will remain same as parent in case of struct column
        num_child_rows[child_col_idx] = num_rows;
      } else {
        parent_column_data[child_col_idx] = {nullptr, 0};
      }
    }
    index += p_col.num_children;
  });
}

std::string get_map_child_col_name(size_t const idx) { return (idx == 0) ? "key" : "value"; }

std::unique_ptr<column> reader::impl::create_empty_column(const size_type orc_col_id,
                                                          column_name_info& schema_info,
                                                          rmm::cuda_stream_view stream)
{
  schema_info.name = _metadata.column_name(0, orc_col_id);
  auto const type  = to_type_id(_metadata.get_schema(orc_col_id),
                               _use_np_dtypes,
                               _timestamp_type.id(),
                               decimal_column_type(decimal128_columns, _metadata, orc_col_id));
  int32_t scale    = 0;
  std::vector<std::unique_ptr<column>> child_columns;
  std::unique_ptr<column> out_col = nullptr;
  auto kind                       = _metadata.get_col_type(orc_col_id).kind;

  switch (kind) {
    case orc::LIST:
      schema_info.children.emplace_back("offsets");
      schema_info.children.emplace_back("");
      out_col = make_lists_column(
        0,
        make_empty_column(type_id::INT32),
        create_empty_column(
          _metadata.get_col_type(orc_col_id).subtypes[0], schema_info.children.back(), stream),
        0,
        rmm::device_buffer{0, stream},
        stream);
      break;
    case orc::MAP: {
      schema_info.children.emplace_back("offsets");
      schema_info.children.emplace_back("struct");
      const auto child_column_ids = _metadata.get_col_type(orc_col_id).subtypes;
      for (size_t idx = 0; idx < _metadata.get_col_type(orc_col_id).subtypes.size(); idx++) {
        auto& children_schema = schema_info.children.back().children;
        children_schema.emplace_back("");
        child_columns.push_back(create_empty_column(
          child_column_ids[idx], schema_info.children.back().children.back(), stream));
        auto name                 = get_map_child_col_name(idx);
        children_schema[idx].name = name;
      }
      auto struct_col =
        make_structs_column(0, std::move(child_columns), 0, rmm::device_buffer{0, stream}, stream);
      out_col = make_lists_column(0,
                                  make_empty_column(type_id::INT32),
                                  std::move(struct_col),
                                  0,
                                  rmm::device_buffer{0, stream},
                                  stream);
    } break;

    case orc::STRUCT:
      for (const auto col : _metadata.get_col_type(orc_col_id).subtypes) {
        schema_info.children.emplace_back("");
        child_columns.push_back(create_empty_column(col, schema_info.children.back(), stream));
      }
      out_col =
        make_structs_column(0, std::move(child_columns), 0, rmm::device_buffer{0, stream}, stream);
      break;

    case orc::DECIMAL:
      if (type == type_id::DECIMAL32 or type == type_id::DECIMAL64 or type == type_id::DECIMAL128) {
        scale = -static_cast<int32_t>(_metadata.get_types()[orc_col_id].scale.value_or(0));
      }
      out_col = make_empty_column(data_type(type, scale));
      break;

    default: out_col = make_empty_column(type);
  }

  return out_col;
}

// Adds child column buffers to parent column
column_buffer&& reader::impl::assemble_buffer(const size_type orc_col_id,
                                              std::vector<std::vector<column_buffer>>& col_buffers,
                                              const size_t level,
                                              rmm::cuda_stream_view stream)
{
  auto const col_id = _col_meta.orc_col_map[level][orc_col_id];
  auto& col_buffer  = col_buffers[level][col_id];

  col_buffer.name = _metadata.column_name(0, orc_col_id);
  auto kind       = _metadata.get_col_type(orc_col_id).kind;
  switch (kind) {
    case orc::LIST:
    case orc::STRUCT:
      for (auto const& col : selected_columns.children[orc_col_id]) {
        col_buffer.children.emplace_back(assemble_buffer(col, col_buffers, level + 1, stream));
      }

      break;
    case orc::MAP: {
      std::vector<column_buffer> child_col_buffers;
      // Get child buffers
      for (size_t idx = 0; idx < selected_columns.children[orc_col_id].size(); idx++) {
        auto name = get_map_child_col_name(idx);
        auto col  = selected_columns.children[orc_col_id][idx];
        child_col_buffers.emplace_back(assemble_buffer(col, col_buffers, level + 1, stream));
        child_col_buffers.back().name = name;
      }
      // Create a struct buffer
      auto num_rows = child_col_buffers[0].size;
      auto struct_buffer =
        column_buffer(cudf::data_type(type_id::STRUCT), num_rows, false, stream, _mr);
      struct_buffer.children = std::move(child_col_buffers);
      struct_buffer.name     = "struct";

      col_buffer.children.emplace_back(std::move(struct_buffer));
    } break;

    default: break;
  }

  return std::move(col_buffer);
}

// creates columns along with schema information for each column
void reader::impl::create_columns(std::vector<std::vector<column_buffer>>&& col_buffers,
                                  std::vector<std::unique_ptr<column>>& out_columns,
                                  std::vector<column_name_info>& schema_info,
                                  rmm::cuda_stream_view stream)
{
  std::transform(selected_columns.levels[0].begin(),
                 selected_columns.levels[0].end(),
                 std::back_inserter(out_columns),
                 [&](auto const col_meta) {
                   schema_info.emplace_back("");
                   auto col_buffer = assemble_buffer(col_meta.id, col_buffers, 0, stream);
                   return make_column(col_buffer, &schema_info.back(), std::nullopt, stream);
                 });
}

reader::impl::impl(std::vector<std::unique_ptr<datasource>>&& sources,
                   orc_reader_options const& options,
                   rmm::cuda_stream_view stream,
                   rmm::mr::device_memory_resource* mr)
  : _mr(mr),
    _sources(std::move(sources)),
    _metadata{_sources, stream},
    selected_columns{_metadata.select_columns(options.get_columns())}
{
  // Override output timestamp resolution if requested
  if (options.get_timestamp_type().id() != type_id::EMPTY) {
    _timestamp_type = options.get_timestamp_type();
  }

  // Enable or disable attempt to use row index for parsing
  _use_index = options.is_enabled_use_index();

  // Enable or disable the conversion to numpy-compatible dtypes
  _use_np_dtypes = options.is_enabled_use_np_dtypes();

  // Control decimals conversion
  decimal128_columns = options.get_decimal128_columns();
}

timezone_table reader::impl::compute_timezone_table(
  const std::vector<cudf::io::orc::metadata::stripe_source_mapping>& selected_stripes,
  rmm::cuda_stream_view stream)
{
  if (selected_stripes.empty()) return {};

  auto const has_timestamp_column = std::any_of(
    selected_columns.levels.cbegin(), selected_columns.levels.cend(), [&](auto& col_lvl) {
      return std::any_of(col_lvl.cbegin(), col_lvl.cend(), [&](auto& col_meta) {
        return _metadata.get_col_type(col_meta.id).kind == TypeKind::TIMESTAMP;
      });
    });
  if (not has_timestamp_column) return {};

  return build_timezone_transition_table(selected_stripes[0].stripe_info[0].second->writerTimezone,
                                         stream);
}

table_with_metadata reader::impl::read(size_type skip_rows,
                                       size_type num_rows,
                                       const std::vector<std::vector<size_type>>& stripes,
                                       rmm::cuda_stream_view stream)
{
  // Selected columns at different levels of nesting are stored in different elements
  // of `selected_columns`; thus, size == 1 means no nested columns
  CUDF_EXPECTS(skip_rows == 0 or selected_columns.num_levels() == 1,
               "skip_rows is not supported by nested columns");

  std::vector<std::unique_ptr<column>> out_columns;
  // buffer and stripe data are stored as per nesting level
  std::vector<std::vector<column_buffer>> out_buffers(selected_columns.num_levels());
  std::vector<column_name_info> schema_info;
  std::vector<std::vector<rmm::device_buffer>> lvl_stripe_data(selected_columns.num_levels());
  std::vector<std::vector<rmm::device_uvector<uint32_t>>> null_count_prefix_sums;
  table_metadata out_metadata;

  // There are no columns in the table
  if (selected_columns.num_levels() == 0)
    return {std::make_unique<table>(), std::move(out_metadata)};

  // Select only stripes required (aka row groups)
  const auto selected_stripes = _metadata.select_stripes(stripes, skip_rows, num_rows, stream);

  auto const tz_table = compute_timezone_table(selected_stripes, stream);

  // Iterates through levels of nested columns, child column will be one level down
  // compared to parent column.
  for (size_t level = 0; level < selected_columns.num_levels(); level++) {
    auto& columns_level = selected_columns.levels[level];
    // Association between each ORC column and its cudf::column
    _col_meta.orc_col_map.emplace_back(_metadata.get_num_cols(), -1);
    std::vector<orc_column_meta> nested_col;

    // Get a list of column data types
    std::vector<data_type> column_types;
    for (auto& col : columns_level) {
      auto col_type = to_type_id(_metadata.get_col_type(col.id),
                                 _use_np_dtypes,
                                 _timestamp_type.id(),
                                 decimal_column_type(decimal128_columns, _metadata, col.id));
      CUDF_EXPECTS(col_type != type_id::EMPTY, "Unknown type");
      if (col_type == type_id::DECIMAL32 or col_type == type_id::DECIMAL64 or
          col_type == type_id::DECIMAL128) {
        // sign of the scale is changed since cuDF follows c++ libraries like CNL
        // which uses negative scaling, but liborc and other libraries
        // follow positive scaling.
        auto const scale =
          -static_cast<size_type>(_metadata.get_col_type(col.id).scale.value_or(0));
        column_types.emplace_back(col_type, scale);
      } else {
        column_types.emplace_back(col_type);
      }

      // Map each ORC column to its column
      _col_meta.orc_col_map[level][col.id] = column_types.size() - 1;
      if (col_type == type_id::LIST or col_type == type_id::STRUCT) nested_col.emplace_back(col);
    }

    // If no rows or stripes to read, return empty columns
    if (num_rows <= 0 || selected_stripes.empty()) {
      std::transform(selected_columns.levels[0].begin(),
                     selected_columns.levels[0].end(),
                     std::back_inserter(out_columns),
                     [&](auto const col_meta) {
                       schema_info.emplace_back("");
                       return create_empty_column(col_meta.id, schema_info.back(), stream);
                     });
      break;
    } else {
      // Get the total number of stripes across all input files.
      size_t total_num_stripes =
        std::accumulate(selected_stripes.begin(),
                        selected_stripes.end(),
                        0,
                        [](size_t sum, auto& stripe_source_mapping) {
                          return sum + stripe_source_mapping.stripe_info.size();
                        });
      const auto num_columns = columns_level.size();
      cudf::detail::hostdevice_2dvector<gpu::ColumnDesc> chunks(
        total_num_stripes, num_columns, stream);
      memset(chunks.base_host_ptr(), 0, chunks.memory_size());

      const bool use_index =
        _use_index &&
        // Do stripes have row group index
        _metadata.is_row_grp_idx_present() &&
        // Only use if we don't have much work with complete columns & stripes
        // TODO: Consider nrows, gpu, and tune the threshold
        (num_rows > _metadata.get_row_index_stride() && !(_metadata.get_row_index_stride() & 7) &&
         _metadata.get_row_index_stride() > 0 && num_columns * total_num_stripes < 8 * 128) &&
        // Only use if first row is aligned to a stripe boundary
        // TODO: Fix logic to handle unaligned rows
        (skip_rows == 0);

      // Logically view streams as columns
      std::vector<orc_stream_info> stream_info;

      null_count_prefix_sums.emplace_back();
      null_count_prefix_sums.back().reserve(selected_columns.levels[level].size());
      std::generate_n(std::back_inserter(null_count_prefix_sums.back()),
                      selected_columns.levels[level].size(),
                      [&]() {
                        return cudf::detail::make_zeroed_device_uvector_async<uint32_t>(
                          total_num_stripes, stream);
                      });

      // Tracker for eventually deallocating compressed and uncompressed data
      auto& stripe_data = lvl_stripe_data[level];

      size_t stripe_start_row = 0;
      size_t num_dict_entries = 0;
      size_t num_rowgroups    = 0;
      int stripe_idx          = 0;

      bool is_level_data_empty = true;
      std::vector<std::pair<std::future<size_t>, size_t>> read_tasks;
      for (auto const& stripe_source_mapping : selected_stripes) {
        // Iterate through the source files selected stripes
        for (auto const& stripe : stripe_source_mapping.stripe_info) {
          const auto stripe_info   = stripe.first;
          const auto stripe_footer = stripe.second;

          auto stream_count          = stream_info.size();
          const auto total_data_size = gather_stream_info(stripe_idx,
                                                          stripe_info,
                                                          stripe_footer,
                                                          _col_meta.orc_col_map[level],
                                                          _metadata.get_types(),
                                                          use_index,
                                                          &num_dict_entries,
                                                          chunks,
                                                          stream_info,
                                                          level == 0);

          auto const is_stripe_data_empty = total_data_size == 0;
          if (not is_stripe_data_empty) { is_level_data_empty = false; }
          CUDF_EXPECTS(not is_stripe_data_empty or stripe_info->indexLength == 0,
                       "Invalid index rowgroup stream data");

          stripe_data.emplace_back(total_data_size, stream);
          auto dst_base = static_cast<uint8_t*>(stripe_data.back().data());

          // Coalesce consecutive streams into one read
          while (not is_stripe_data_empty and stream_count < stream_info.size()) {
            const auto d_dst  = dst_base + stream_info[stream_count].dst_pos;
            const auto offset = stream_info[stream_count].offset;
            auto len          = stream_info[stream_count].length;
            stream_count++;

            while (stream_count < stream_info.size() &&
                   stream_info[stream_count].offset == offset + len) {
              len += stream_info[stream_count].length;
              stream_count++;
            }
            if (_metadata.per_file_metadata[stripe_source_mapping.source_idx]
                  .source->is_device_read_preferred(len)) {
              read_tasks.push_back(
                std::pair(_metadata.per_file_metadata[stripe_source_mapping.source_idx]
                            .source->device_read_async(offset, len, d_dst, stream),
                          len));

            } else {
              const auto buffer =
                _metadata.per_file_metadata[stripe_source_mapping.source_idx].source->host_read(
                  offset, len);
              CUDF_EXPECTS(buffer->size() == len, "Unexpected discrepancy in bytes read.");
              CUDF_CUDA_TRY(hipMemcpyAsync(
                d_dst, buffer->data(), len, hipMemcpyHostToDevice, stream.value()));
              stream.synchronize();
            }
          }

          const auto num_rows_per_stripe = stripe_info->numberOfRows;
          const auto rowgroup_id         = num_rowgroups;
          auto stripe_num_rowgroups      = 0;
          if (use_index) {
            stripe_num_rowgroups = (num_rows_per_stripe + _metadata.get_row_index_stride() - 1) /
                                   _metadata.get_row_index_stride();
          }
          // Update chunks to reference streams pointers
          for (size_t col_idx = 0; col_idx < num_columns; col_idx++) {
            auto& chunk = chunks[stripe_idx][col_idx];
            // start row, number of rows in a each stripe and total number of rows
            // may change in lower levels of nesting
            chunk.start_row = (level == 0)
                                ? stripe_start_row
                                : _col_meta.child_start_row[stripe_idx * num_columns + col_idx];
            chunk.num_rows =
              (level == 0)
                ? stripe_info->numberOfRows
                : _col_meta.num_child_rows_per_stripe[stripe_idx * num_columns + col_idx];
            chunk.column_num_rows = (level == 0) ? num_rows : _col_meta.num_child_rows[col_idx];
            chunk.parent_validity_info =
              (level == 0) ? column_validity_info{} : _col_meta.parent_column_data[col_idx];
            chunk.parent_null_count_prefix_sums =
              (level == 0)
                ? nullptr
                : null_count_prefix_sums[level - 1][_col_meta.parent_column_index[col_idx]].data();
            chunk.encoding_kind = stripe_footer->columns[columns_level[col_idx].id].kind;
            chunk.type_kind     = _metadata.per_file_metadata[stripe_source_mapping.source_idx]
                                .ff.types[columns_level[col_idx].id]
                                .kind;
            // num_child_rows for a struct column will be same, for other nested types it will be
            // calculated.
            chunk.num_child_rows = (chunk.type_kind != orc::STRUCT) ? 0 : chunk.num_rows;
            chunk.dtype_id       = column_types[col_idx].id();
            chunk.decimal_scale  = _metadata.per_file_metadata[stripe_source_mapping.source_idx]
                                    .ff.types[columns_level[col_idx].id]
                                    .scale.value_or(0);

            chunk.rowgroup_id   = rowgroup_id;
            chunk.dtype_len     = (column_types[col_idx].id() == type_id::STRING)
                                    ? sizeof(string_index_pair)
                                  : ((column_types[col_idx].id() == type_id::LIST) or
                                 (column_types[col_idx].id() == type_id::STRUCT))
                                    ? sizeof(size_type)
                                    : cudf::size_of(column_types[col_idx]);
            chunk.num_rowgroups = stripe_num_rowgroups;
            if (chunk.type_kind == orc::TIMESTAMP) {
              chunk.timestamp_type_id = _timestamp_type.id();
            }
            if (not is_stripe_data_empty) {
              for (int k = 0; k < gpu::CI_NUM_STREAMS; k++) {
                chunk.streams[k] = dst_base + stream_info[chunk.strm_id[k]].dst_pos;
              }
            }
          }
          stripe_start_row += num_rows_per_stripe;
          num_rowgroups += stripe_num_rowgroups;

          stripe_idx++;
        }
      }
      for (auto& task : read_tasks) {
        CUDF_EXPECTS(task.first.get() == task.second, "Unexpected discrepancy in bytes read.");
      }

      // Process dataset chunk pages into output columns
      if (stripe_data.size() != 0) {
        auto row_groups =
          cudf::detail::hostdevice_2dvector<gpu::RowGroup>(num_rowgroups, num_columns, stream);
        if (level > 0 and row_groups.size().first) {
          cudf::host_span<gpu::RowGroup> row_groups_span(row_groups.base_host_ptr(),
                                                         num_rowgroups * num_columns);
          auto& rw_grp_meta = _col_meta.rwgrp_meta;

          // Update start row and num rows per row group
          std::transform(rw_grp_meta.begin(),
                         rw_grp_meta.end(),
                         row_groups_span.begin(),
                         rw_grp_meta.begin(),
                         [&](auto meta, auto& row_grp) {
                           row_grp.num_rows  = meta.num_rows;
                           row_grp.start_row = meta.start_row;
                           return meta;
                         });
        }
        // Setup row group descriptors if using indexes
        if (_metadata.per_file_metadata[0].ps.compression != orc::NONE and
            not is_level_data_empty) {
          auto decomp_data = decompress_stripe_data(chunks,
                                                    stripe_data,
                                                    *_metadata.per_file_metadata[0].decompressor,
                                                    stream_info,
                                                    total_num_stripes,
                                                    row_groups,
                                                    _metadata.get_row_index_stride(),
                                                    level == 0,
                                                    stream);
          stripe_data.clear();
          stripe_data.push_back(std::move(decomp_data));
        } else {
          if (row_groups.size().first) {
            chunks.host_to_device(stream);
            row_groups.host_to_device(stream);
            gpu::ParseRowGroupIndex(row_groups.base_device_ptr(),
                                    nullptr,
                                    chunks.base_device_ptr(),
                                    num_columns,
                                    total_num_stripes,
                                    num_rowgroups,
                                    _metadata.get_row_index_stride(),
                                    level == 0,
                                    stream);
          }
        }

        for (size_t i = 0; i < column_types.size(); ++i) {
          bool is_nullable = false;
          for (size_t j = 0; j < total_num_stripes; ++j) {
            if (chunks[j][i].strm_len[gpu::CI_PRESENT] != 0) {
              is_nullable = true;
              break;
            }
          }
          auto is_list_type = (column_types[i].id() == type_id::LIST);
          auto n_rows       = (level == 0) ? num_rows : _col_meta.num_child_rows[i];
          // For list column, offset column will be always size + 1
          if (is_list_type) n_rows++;
          out_buffers[level].emplace_back(column_types[i], n_rows, is_nullable, stream, _mr);
        }

        if (not is_level_data_empty) {
          decode_stream_data(chunks,
                             num_dict_entries,
                             skip_rows,
                             tz_table.view(),
                             row_groups,
                             _metadata.get_row_index_stride(),
                             out_buffers[level],
                             level,
                             stream);
        }

        // Extract information to process nested child columns
        if (nested_col.size()) {
          if (not is_level_data_empty) {
            scan_null_counts(chunks, null_count_prefix_sums[level], stream);
          }
          row_groups.device_to_host(stream, true);
          aggregate_child_meta(chunks, row_groups, out_buffers[level], nested_col, level);
        }

        // ORC stores number of elements at each row, so we need to generate offsets from that
        if (nested_col.size()) {
          std::vector<list_buffer_data> buff_data;
          std::for_each(
            out_buffers[level].begin(), out_buffers[level].end(), [&buff_data](auto& out_buffer) {
              if (out_buffer.type.id() == type_id::LIST) {
                auto data = static_cast<size_type*>(out_buffer.data());
                buff_data.emplace_back(list_buffer_data{data, out_buffer.size});
              }
            });

          if (buff_data.size()) {
            auto const dev_buff_data = cudf::detail::make_device_uvector_async(buff_data, stream);
            generate_offsets_for_list(dev_buff_data, stream);
          }
        }
      }
    }
  }

  // If out_columns is empty, then create columns from buffer.
  if (out_columns.empty()) {
    create_columns(std::move(out_buffers), out_columns, schema_info, stream);
  }

  // Return column names (must match order of returned columns)
  out_metadata.column_names.reserve(schema_info.size());
  std::transform(schema_info.cbegin(),
                 schema_info.cend(),
                 std::back_inserter(out_metadata.column_names),
                 [](auto info) { return info.name; });

  out_metadata.schema_info = std::move(schema_info);

  std::transform(_metadata.per_file_metadata.cbegin(),
                 _metadata.per_file_metadata.cend(),
                 std::back_inserter(out_metadata.per_file_user_data),
                 [](auto& meta) {
                   std::unordered_map<std::string, std::string> kv_map;
                   std::transform(meta.ff.metadata.cbegin(),
                                  meta.ff.metadata.cend(),
                                  std::inserter(kv_map, kv_map.end()),
                                  [](auto const& kv) {
                                    return std::pair{kv.name, kv.value};
                                  });
                   return kv_map;
                 });
  out_metadata.user_data = {out_metadata.per_file_user_data[0].begin(),
                            out_metadata.per_file_user_data[0].end()};

  return {std::make_unique<table>(std::move(out_columns)), std::move(out_metadata)};
}

// Forward to implementation
reader::reader(std::vector<std::unique_ptr<cudf::io::datasource>>&& sources,
               orc_reader_options const& options,
               rmm::cuda_stream_view stream,
               rmm::mr::device_memory_resource* mr)
{
  _impl = std::make_unique<impl>(std::move(sources), options, stream, mr);
}

// Destructor within this translation unit
reader::~reader() = default;

// Forward to implementation
table_with_metadata reader::read(orc_reader_options const& options, rmm::cuda_stream_view stream)
{
  return _impl->read(
    options.get_skip_rows(), options.get_num_rows(), options.get_stripes(), stream);
}

}  // namespace orc
}  // namespace detail
}  // namespace io
}  // namespace cudf
