#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019-2021, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

/**
 * @file reader_impl.cu
 * @brief cuDF-IO ORC reader class implementation
 */

#include "io/orc/orc_gpu.h"
#include "reader_impl.hpp"
#include "timezone.cuh"

#include <io/comp/gpuinflate.h>
#include "orc.h"

#include <cudf/table/table.hpp>
#include <cudf/utilities/error.hpp>
#include <cudf/utilities/traits.hpp>

#include <iterator>
#include <rmm/cuda_stream_view.hpp>
#include <rmm/device_buffer.hpp>
#include <rmm/device_uvector.hpp>

#include <algorithm>
#include <array>

namespace cudf {
namespace io {
namespace detail {
namespace orc {
// Import functionality that's independent of legacy code
using namespace cudf::io::orc;
using namespace cudf::io;

namespace {
/**
 * @brief Function that translates ORC data kind to cuDF type enum
 */
constexpr type_id to_type_id(const orc::SchemaType &schema,
                             bool use_np_dtypes,
                             type_id timestamp_type_id,
                             bool decimals_as_float64)
{
  switch (schema.kind) {
    case orc::BOOLEAN: return type_id::BOOL8;
    case orc::BYTE: return type_id::INT8;
    case orc::SHORT: return type_id::INT16;
    case orc::INT: return type_id::INT32;
    case orc::LONG: return type_id::INT64;
    case orc::FLOAT: return type_id::FLOAT32;
    case orc::DOUBLE: return type_id::FLOAT64;
    case orc::STRING:
    case orc::BINARY:
    case orc::VARCHAR:
    case orc::CHAR:
      // Variable-length types can all be mapped to STRING
      return type_id::STRING;
    case orc::TIMESTAMP:
      return (timestamp_type_id != type_id::EMPTY) ? timestamp_type_id
                                                   : type_id::TIMESTAMP_NANOSECONDS;
    case orc::DATE:
      // There isn't a (DAYS -> np.dtype) mapping
      return (use_np_dtypes) ? type_id::TIMESTAMP_MILLISECONDS : type_id::TIMESTAMP_DAYS;
    case orc::DECIMAL: return (decimals_as_float64) ? type_id::FLOAT64 : type_id::DECIMAL64;
    default: break;
  }

  return type_id::EMPTY;
}

/**
 * @brief Function that translates cuDF time unit to ORC clock frequency
 */
constexpr int32_t to_clockrate(type_id timestamp_type_id)
{
  switch (timestamp_type_id) {
    case type_id::TIMESTAMP_SECONDS: return 1;
    case type_id::TIMESTAMP_MILLISECONDS: return 1000;
    case type_id::TIMESTAMP_MICROSECONDS: return 1000000;
    case type_id::TIMESTAMP_NANOSECONDS: return 1000000000;
    default: return 0;
  }
}

constexpr std::pair<gpu::StreamIndexType, uint32_t> get_index_type_and_pos(
  const orc::StreamKind kind, uint32_t skip_count, bool non_child)
{
  switch (kind) {
    case orc::DATA:
      skip_count += 1;
      skip_count |= (skip_count & 0xff) << 8;
      return std::make_pair(gpu::CI_DATA, skip_count);
    case orc::LENGTH:
    case orc::SECONDARY:
      skip_count += 1;
      skip_count |= (skip_count & 0xff) << 16;
      return std::make_pair(gpu::CI_DATA2, skip_count);
    case orc::DICTIONARY_DATA: return std::make_pair(gpu::CI_DICTIONARY, skip_count);
    case orc::PRESENT:
      skip_count += (non_child ? 1 : 0);
      return std::make_pair(gpu::CI_PRESENT, skip_count);
    case orc::ROW_INDEX: return std::make_pair(gpu::CI_INDEX, skip_count);
    default:
      // Skip this stream as it's not strictly required
      return std::make_pair(gpu::CI_NUM_STREAMS, 0);
  }
}

}  // namespace

namespace {
/**
 * @brief Struct that maps ORC streams to columns
 */
struct orc_stream_info {
  orc_stream_info() = default;
  explicit orc_stream_info(
    uint64_t offset_, size_t dst_pos_, uint32_t length_, uint32_t gdf_idx_, uint32_t stripe_idx_)
    : offset(offset_),
      dst_pos(dst_pos_),
      length(length_),
      gdf_idx(gdf_idx_),
      stripe_idx(stripe_idx_)
  {
  }
  uint64_t offset;      // offset in file
  size_t dst_pos;       // offset in memory relative to start of compressed stripe data
  size_t length;        // length in file
  uint32_t gdf_idx;     // column index
  uint32_t stripe_idx;  // stripe index
};

/**
 * @brief Function that populates column descriptors stream/chunk
 */
size_t gather_stream_info(const size_t stripe_index,
                          const orc::StripeInformation *stripeinfo,
                          const orc::StripeFooter *stripefooter,
                          const std::vector<int> &orc2gdf,
                          const std::vector<int> &gdf2orc,
                          const std::vector<orc::SchemaType> types,
                          bool use_index,
                          size_t *num_dictionary_entries,
                          hostdevice_vector<gpu::ColumnDesc> &chunks,
                          std::vector<orc_stream_info> &stream_info)
{
  const auto num_columns = gdf2orc.size();
  uint64_t src_offset    = 0;
  uint64_t dst_offset    = 0;
  for (const auto &stream : stripefooter->streams) {
    if (!stream.column_id || *stream.column_id >= orc2gdf.size()) {
      dst_offset += stream.length;
      continue;
    }

    auto const column_id = *stream.column_id;
    auto col             = orc2gdf[column_id];

    if (col == -1) {
      // A struct-type column has no data itself, but rather child columns
      // for each of its fields. There is only a PRESENT stream, which
      // needs to be included for the reader.
      const auto schema_type = types[column_id];
      if (schema_type.subtypes.size() != 0) {
        if (schema_type.kind == orc::STRUCT && stream.kind == orc::PRESENT) {
          for (const auto &idx : schema_type.subtypes) {
            auto child_idx = (idx < orc2gdf.size()) ? orc2gdf[idx] : -1;
            if (child_idx >= 0) {
              col                             = child_idx;
              auto &chunk                     = chunks[stripe_index * num_columns + col];
              chunk.strm_id[gpu::CI_PRESENT]  = stream_info.size();
              chunk.strm_len[gpu::CI_PRESENT] = stream.length;
            }
          }
        }
      }
    }
    if (col != -1) {
      if (src_offset >= stripeinfo->indexLength || use_index) {
        // NOTE: skip_count field is temporarily used to track index ordering
        auto &chunk = chunks[stripe_index * num_columns + col];
        const auto idx =
          get_index_type_and_pos(stream.kind, chunk.skip_count, col == orc2gdf[column_id]);
        if (idx.first < gpu::CI_NUM_STREAMS) {
          chunk.strm_id[idx.first]  = stream_info.size();
          chunk.strm_len[idx.first] = stream.length;
          chunk.skip_count          = idx.second;

          if (idx.first == gpu::CI_DICTIONARY) {
            chunk.dictionary_start = *num_dictionary_entries;
            chunk.dict_len         = stripefooter->columns[column_id].dictionarySize;
            *num_dictionary_entries += stripefooter->columns[column_id].dictionarySize;
          }
        }
      }
      stream_info.emplace_back(
        stripeinfo->offset + src_offset, dst_offset, stream.length, col, stripe_index);
      dst_offset += stream.length;
    }
    src_offset += stream.length;
  }

  return dst_offset;
}

/**
 * @brief Determines if a column should be converted from decimal to float
 */
bool should_convert_decimal_column_to_float(const std::vector<std::string> &columns_to_convert,
                                            cudf::io::orc::metadata &metadata,
                                            int column_index)
{
  return (std::find(columns_to_convert.begin(),
                    columns_to_convert.end(),
                    metadata.get_column_name(column_index)) != columns_to_convert.end());
}

}  // namespace

/**
 * @brief In order to support multiple input files/buffers we need to gather
 * the metadata across all of those input(s). This class provides a place
 * to aggregate that metadata from all the files.
 */
class aggregate_orc_metadata {
  using OrcStripeInfo = std::pair<const StripeInformation *, const StripeFooter *>;

 public:
  mutable std::vector<cudf::io::orc::metadata> per_file_metadata;
  size_type const num_rows;
  size_type const num_columns;
  size_type const num_stripes;

  /**
   * @brief Create a metadata object from each element in the source vector
   */
  auto metadatas_from_sources(std::vector<std::unique_ptr<datasource>> const &sources)
  {
    std::vector<cudf::io::orc::metadata> metadatas;
    std::transform(
      sources.cbegin(), sources.cend(), std::back_inserter(metadatas), [](auto const &source) {
        return cudf::io::orc::metadata(source.get());
      });
    return metadatas;
  }

  /**
   * @brief Sums up the number of rows of each source
   */
  size_type calc_num_rows() const
  {
    return std::accumulate(
      per_file_metadata.begin(), per_file_metadata.end(), 0, [](auto &sum, auto &pfm) {
        return sum + pfm.get_total_rows();
      });
  }

  /**
   * @brief Number of columns in a ORC file.
   */
  size_type calc_num_cols() const
  {
    if (not per_file_metadata.empty()) { return per_file_metadata[0].get_num_columns(); }
    return 0;
  }

  /**
   * @brief Sums up the number of stripes of each source
   */
  size_type calc_num_stripes() const
  {
    return std::accumulate(
      per_file_metadata.begin(), per_file_metadata.end(), 0, [](auto &sum, auto &pfm) {
        return sum + pfm.get_num_stripes();
      });
  }

  aggregate_orc_metadata(std::vector<std::unique_ptr<datasource>> const &sources)
    : per_file_metadata(metadatas_from_sources(sources)),
      num_rows(calc_num_rows()),
      num_columns(calc_num_cols()),
      num_stripes(calc_num_stripes())
  {
    // Verify that the input files have the same number of columns,
    // as well as matching types, compression, and names
    for (auto const &pfm : per_file_metadata) {
      CUDF_EXPECTS(per_file_metadata[0].get_num_columns() == pfm.get_num_columns(),
                   "All sources must have the same number of columns");
      CUDF_EXPECTS(per_file_metadata[0].ps.compression == pfm.ps.compression,
                   "All sources must have the same compression type");

      // Check the types, column names, and decimal scale
      for (size_t i = 0; i < pfm.ff.types.size(); i++) {
        CUDF_EXPECTS(pfm.ff.types[i].kind == per_file_metadata[0].ff.types[i].kind,
                     "Column types across all input sources must be the same");
        CUDF_EXPECTS(std::equal(pfm.ff.types[i].fieldNames.begin(),
                                pfm.ff.types[i].fieldNames.end(),
                                per_file_metadata[0].ff.types[i].fieldNames.begin()),
                     "All source column names must be the same");
        CUDF_EXPECTS(
          pfm.ff.types[i].scale.value_or(0) == per_file_metadata[0].ff.types[i].scale.value_or(0),
          "All scale values must be the same");
      }
    }
  }

  auto const &get_schema(int schema_idx) const { return per_file_metadata[0].ff.types[schema_idx]; }

  auto get_col_type(int col_idx) const { return per_file_metadata[0].ff.types[col_idx]; }

  auto get_num_rows() const { return num_rows; }

  auto get_num_cols() const { return per_file_metadata[0].get_num_columns(); }

  auto get_num_stripes() const { return num_stripes; }

  auto get_num_source_files() const { return per_file_metadata.size(); }

  auto const &get_types() const { return per_file_metadata[0].ff.types; }

  int get_row_index_stride() const { return per_file_metadata[0].ff.rowIndexStride; }

  auto get_column_name(const int source_idx, const int column_idx) const
  {
    CUDF_EXPECTS(source_idx <= static_cast<int>(per_file_metadata.size()),
                 "Out of range source_idx provided");
    CUDF_EXPECTS(column_idx <= per_file_metadata[source_idx].get_num_columns(),
                 "Out of range column_idx provided");
    return per_file_metadata[source_idx].get_column_name(column_idx);
  }

  std::vector<cudf::io::orc::metadata::stripe_source_mapping> select_stripes(
    std::vector<std::vector<size_type>> const &user_specified_stripes,
    size_type &row_start,
    size_type &row_count)
  {
    std::vector<cudf::io::orc::metadata::stripe_source_mapping> selected_stripes_mapping;

    if (!user_specified_stripes.empty()) {
      CUDF_EXPECTS(user_specified_stripes.size() == get_num_source_files(),
                   "Must specify stripes for each source");
      // row_start is 0 if stripes are set. If this is not true anymore, then
      // row_start needs to be subtracted to get the correct row_count
      CUDF_EXPECTS(row_start == 0, "Start row index should be 0");

      row_count = 0;
      // Each vector entry represents a source file; each nested vector represents the
      // user_defined_stripes to get from that source file
      for (size_t src_file_idx = 0; src_file_idx < user_specified_stripes.size(); ++src_file_idx) {
        std::vector<OrcStripeInfo> stripe_infos;

        // Coalesce stripe info at the source file later since that makes downstream processing much
        // easier in impl::read
        for (const size_t &stripe_idx : user_specified_stripes[src_file_idx]) {
          CUDF_EXPECTS(stripe_idx < per_file_metadata[src_file_idx].ff.stripes.size(),
                       "Invalid stripe index");
          stripe_infos.push_back(
            std::make_pair(&per_file_metadata[src_file_idx].ff.stripes[stripe_idx], nullptr));
          row_count += per_file_metadata[src_file_idx].ff.stripes[stripe_idx].numberOfRows;
        }
        selected_stripes_mapping.push_back({static_cast<int>(src_file_idx), stripe_infos});
      }
    } else {
      row_start = std::max(row_start, 0);
      if (row_count < 0) {
        row_count = static_cast<size_type>(
          std::min<int64_t>(get_num_rows(), std::numeric_limits<size_type>::max()));
      }
      row_count = std::min(row_count, get_num_rows() - row_start);
      CUDF_EXPECTS(row_count >= 0, "Invalid row count");
      CUDF_EXPECTS(row_start <= get_num_rows(), "Invalid row start");

      size_type count = 0;
      // Iterate all source files, each source file has corelating metadata
      for (size_t src_file_idx = 0;
           src_file_idx < per_file_metadata.size() && count < row_start + row_count;
           ++src_file_idx) {
        std::vector<OrcStripeInfo> stripe_infos;

        for (size_t stripe_idx = 0;
             stripe_idx < per_file_metadata[src_file_idx].ff.stripes.size() &&
             count < row_start + row_count;
             ++stripe_idx) {
          count += per_file_metadata[src_file_idx].ff.stripes[stripe_idx].numberOfRows;
          if (count > row_start || count == 0) {
            stripe_infos.push_back(
              std::make_pair(&per_file_metadata[src_file_idx].ff.stripes[stripe_idx], nullptr));
          }
        }

        selected_stripes_mapping.push_back({static_cast<int>(src_file_idx), stripe_infos});
      }
    }

    // Read each stripe's stripefooter metadata
    if (not selected_stripes_mapping.empty()) {
      for (auto &mapping : selected_stripes_mapping) {
        // Resize to all stripe_info for the source level
        per_file_metadata[mapping.source_idx].stripefooters.resize(mapping.stripe_info.size());

        for (size_t i = 0; i < mapping.stripe_info.size(); i++) {
          const auto stripe         = mapping.stripe_info[i].first;
          const auto sf_comp_offset = stripe->offset + stripe->indexLength + stripe->dataLength;
          const auto sf_comp_length = stripe->footerLength;
          CUDF_EXPECTS(
            sf_comp_offset + sf_comp_length < per_file_metadata[mapping.source_idx].source->size(),
            "Invalid stripe information");
          const auto buffer =
            per_file_metadata[mapping.source_idx].source->host_read(sf_comp_offset, sf_comp_length);
          size_t sf_length = 0;
          auto sf_data     = per_file_metadata[mapping.source_idx].decompressor->Decompress(
            buffer->data(), sf_comp_length, &sf_length);
          ProtobufReader(sf_data, sf_length)
            .read(per_file_metadata[mapping.source_idx].stripefooters[i]);
          mapping.stripe_info[i].second = &per_file_metadata[mapping.source_idx].stripefooters[i];
        }
      }
    }

    return selected_stripes_mapping;
  }

  /**
   * @brief Filters and reduces down to a selection of columns
   *
   * @param use_names List of column names to select
   * @param has_timestamp_column True if timestamp column present and false otherwise
   *
   * @return input column information, output column information, list of output column schema
   * indices
   */
  std::vector<int> select_columns(std::vector<std::string> const &use_names,
                                  bool &has_timestamp_column) const
  {
    auto const &pfm = per_file_metadata[0];

    std::vector<int> output_column_schema_idxs;
    if (not use_names.empty()) {
      int index = 0;
      for (auto const &use_name : use_names) {
        bool name_found = false;
        for (int i = 0; i < pfm.get_num_columns(); ++i, ++index) {
          if (index >= pfm.get_num_columns()) { index = 0; }
          if (pfm.get_column_name(index).compare(use_name) == 0) {
            name_found = true;
            output_column_schema_idxs.emplace_back(index);
            if (pfm.ff.types[index].kind == orc::TIMESTAMP) { has_timestamp_column = true; }
            index++;
            break;
          }
        }
        CUDF_EXPECTS(name_found, "Unknown column name : " + std::string(use_name));
      }
    } else {
      // For now, only select all leaf nodes
      for (int i = 1; i < pfm.get_num_columns(); ++i) {
        if (pfm.ff.types[i].subtypes.empty()) {
          output_column_schema_idxs.emplace_back(i);
          if (pfm.ff.types[i].kind == orc::TIMESTAMP) { has_timestamp_column = true; }
        }
      }
    }

    return output_column_schema_idxs;
  }
};

rmm::device_buffer reader::impl::decompress_stripe_data(
  hostdevice_vector<gpu::ColumnDesc> &chunks,
  const std::vector<rmm::device_buffer> &stripe_data,
  const OrcDecompressor *decompressor,
  std::vector<orc_stream_info> &stream_info,
  size_t num_stripes,
  device_span<gpu::RowGroup> row_groups,
  size_t row_index_stride,
  rmm::cuda_stream_view stream)
{
  // Parse the columns' compressed info
  hostdevice_vector<gpu::CompressedStreamInfo> compinfo(0, stream_info.size(), stream);
  for (const auto &info : stream_info) {
    compinfo.insert(gpu::CompressedStreamInfo(
      static_cast<const uint8_t *>(stripe_data[info.stripe_idx].data()) + info.dst_pos,
      info.length));
  }
  compinfo.host_to_device(stream);
  gpu::ParseCompressedStripeData(compinfo.device_ptr(),
                                 compinfo.size(),
                                 decompressor->GetBlockSize(),
                                 decompressor->GetLog2MaxCompressionRatio(),
                                 stream);
  compinfo.device_to_host(stream, true);

  // Count the exact number of compressed blocks
  size_t num_compressed_blocks   = 0;
  size_t num_uncompressed_blocks = 0;
  size_t total_decomp_size       = 0;
  for (size_t i = 0; i < compinfo.size(); ++i) {
    num_compressed_blocks += compinfo[i].num_compressed_blocks;
    num_uncompressed_blocks += compinfo[i].num_uncompressed_blocks;
    total_decomp_size += compinfo[i].max_uncompressed_size;
  }
  CUDF_EXPECTS(total_decomp_size > 0, "No decompressible data found");

  rmm::device_buffer decomp_data(total_decomp_size, stream);
  rmm::device_uvector<gpu_inflate_input_s> inflate_in(
    num_compressed_blocks + num_uncompressed_blocks, stream);
  rmm::device_uvector<gpu_inflate_status_s> inflate_out(num_compressed_blocks, stream);

  // Parse again to populate the decompression input/output buffers
  size_t decomp_offset      = 0;
  uint32_t start_pos        = 0;
  uint32_t start_pos_uncomp = (uint32_t)num_compressed_blocks;
  for (size_t i = 0; i < compinfo.size(); ++i) {
    auto dst_base                 = static_cast<uint8_t *>(decomp_data.data());
    compinfo[i].uncompressed_data = dst_base + decomp_offset;
    compinfo[i].decctl            = inflate_in.data() + start_pos;
    compinfo[i].decstatus         = inflate_out.data() + start_pos;
    compinfo[i].copyctl           = inflate_in.data() + start_pos_uncomp;

    stream_info[i].dst_pos = decomp_offset;
    decomp_offset += compinfo[i].max_uncompressed_size;
    start_pos += compinfo[i].num_compressed_blocks;
    start_pos_uncomp += compinfo[i].num_uncompressed_blocks;
  }
  compinfo.host_to_device(stream);
  gpu::ParseCompressedStripeData(compinfo.device_ptr(),
                                 compinfo.size(),
                                 decompressor->GetBlockSize(),
                                 decompressor->GetLog2MaxCompressionRatio(),
                                 stream);

  // Dispatch batches of blocks to decompress
  if (num_compressed_blocks > 0) {
    switch (decompressor->GetKind()) {
      case orc::ZLIB:
        CUDA_TRY(
          gpuinflate(inflate_in.data(), inflate_out.data(), num_compressed_blocks, 0, stream));
        break;
      case orc::SNAPPY:
        CUDA_TRY(gpu_unsnap(inflate_in.data(), inflate_out.data(), num_compressed_blocks, stream));
        break;
      default: CUDF_EXPECTS(false, "Unexpected decompression dispatch"); break;
    }
  }
  if (num_uncompressed_blocks > 0) {
    CUDA_TRY(gpu_copy_uncompressed_blocks(
      inflate_in.data() + num_compressed_blocks, num_uncompressed_blocks, stream));
  }
  gpu::PostDecompressionReassemble(compinfo.device_ptr(), compinfo.size(), stream);

  // Update the stream information with the updated uncompressed info
  // TBD: We could update the value from the information we already
  // have in stream_info[], but using the gpu results also updates
  // max_uncompressed_size to the actual uncompressed size, or zero if
  // decompression failed.
  compinfo.device_to_host(stream, true);

  const size_t num_columns = chunks.size() / num_stripes;

  for (size_t i = 0; i < num_stripes; ++i) {
    for (size_t j = 0; j < num_columns; ++j) {
      auto &chunk = chunks[i * num_columns + j];
      for (int k = 0; k < gpu::CI_NUM_STREAMS; ++k) {
        if (chunk.strm_len[k] > 0 && chunk.strm_id[k] < compinfo.size()) {
          chunk.streams[k]  = compinfo[chunk.strm_id[k]].uncompressed_data;
          chunk.strm_len[k] = compinfo[chunk.strm_id[k]].max_uncompressed_size;
        }
      }
    }
  }

  if (not row_groups.empty()) {
    chunks.host_to_device(stream);
    gpu::ParseRowGroupIndex(row_groups.data(),
                            compinfo.device_ptr(),
                            chunks.device_ptr(),
                            num_columns,
                            num_stripes,
                            row_groups.size() / num_columns,
                            row_index_stride,
                            stream);
  }

  return decomp_data;
}

void reader::impl::decode_stream_data(hostdevice_vector<gpu::ColumnDesc> &chunks,
                                      size_t num_dicts,
                                      size_t skip_rows,
                                      size_t num_rows,
                                      timezone_table_view tz_table,
                                      device_span<gpu::RowGroup const> row_groups,
                                      size_t row_index_stride,
                                      std::vector<column_buffer> &out_buffers,
                                      rmm::cuda_stream_view stream)
{
  const auto num_columns = out_buffers.size();
  const auto num_stripes = chunks.size() / out_buffers.size();

  // Update chunks with pointers to column data
  for (size_t i = 0; i < num_stripes; ++i) {
    for (size_t j = 0; j < num_columns; ++j) {
      auto &chunk            = chunks[i * num_columns + j];
      chunk.column_data_base = out_buffers[j].data();
      chunk.valid_map_base   = out_buffers[j].null_mask();
    }
  }

  // Allocate global dictionary for deserializing
  rmm::device_uvector<gpu::DictionaryEntry> global_dict(num_dicts, stream);

  chunks.host_to_device(stream);
  gpu::DecodeNullsAndStringDictionaries(
    chunks.device_ptr(), global_dict.data(), num_columns, num_stripes, num_rows, skip_rows, stream);
  gpu::DecodeOrcColumnData(chunks.device_ptr(),
                           global_dict.data(),
                           num_columns,
                           num_stripes,
                           num_rows,
                           skip_rows,
                           tz_table,
                           row_groups.data(),
                           row_groups.size() / num_columns,
                           row_index_stride,
                           stream);
  chunks.device_to_host(stream, true);

  for (size_t i = 0; i < num_stripes; ++i) {
    for (size_t j = 0; j < num_columns; ++j) {
      out_buffers[j].null_count() += chunks[i * num_columns + j].null_count;
    }
  }
}

reader::impl::impl(std::vector<std::unique_ptr<datasource>> &&sources,
                   orc_reader_options const &options,
                   rmm::mr::device_memory_resource *mr)
  : _mr(mr), _sources(std::move(sources))
{
  // Open and parse the source(s) dataset metadata
  _metadata = std::make_unique<aggregate_orc_metadata>(_sources);

  // Select only columns required by the options
  _selected_columns = _metadata->select_columns(options.get_columns(), _has_timestamp_column);

  // Override output timestamp resolution if requested
  if (options.get_timestamp_type().id() != type_id::EMPTY) {
    _timestamp_type = options.get_timestamp_type();
  }

  // Enable or disable attempt to use row index for parsing
  _use_index = options.is_enabled_use_index();

  // Enable or disable the conversion to numpy-compatible dtypes
  _use_np_dtypes = options.is_enabled_use_np_dtypes();

  // Control decimals conversion (float64 or int64 with optional scale)
  _decimal_cols_as_float = options.get_decimal_cols_as_float();
}

table_with_metadata reader::impl::read(size_type skip_rows,
                                       size_type num_rows,
                                       const std::vector<std::vector<size_type>> &stripes,
                                       rmm::cuda_stream_view stream)
{
  std::vector<std::unique_ptr<column>> out_columns;
  table_metadata out_metadata;

  // There are no columns in table
  if (_selected_columns.size() == 0) return {std::make_unique<table>(), std::move(out_metadata)};

  // Select only stripes required (aka row groups)
  const auto selected_stripes = _metadata->select_stripes(stripes, skip_rows, num_rows);

  // Association between each ORC column and its cudf::column
  std::vector<int32_t> orc_col_map(_metadata->get_num_cols(), -1);

  // Get a list of column data types
  std::vector<data_type> column_types;
  for (const auto &col : _selected_columns) {
    // If the column type is orc::DECIMAL see if the user
    // desires it to be converted to float64 or not
    auto const decimal_as_float64 = should_convert_decimal_column_to_float(
      _decimal_cols_as_float, _metadata->per_file_metadata[0], col);

    auto col_type = to_type_id(
      _metadata->get_col_type(col), _use_np_dtypes, _timestamp_type.id(), decimal_as_float64);
    CUDF_EXPECTS(col_type != type_id::EMPTY, "Unknown type");
    // Remove this once we support Decimal128 data type
    CUDF_EXPECTS((col_type != type_id::DECIMAL64) or (_metadata->get_col_type(col).precision <= 18),
                 "Decimal data has precision > 18, Decimal64 data type doesn't support it.");
    if (col_type == type_id::DECIMAL64) {
      // sign of the scale is changed since cuDF follows c++ libraries like CNL
      // which uses negative scaling, but liborc and other libraries
      // follow positive scaling.
      auto const scale = -static_cast<int32_t>(_metadata->get_col_type(col).scale.value_or(0));
      column_types.emplace_back(col_type, scale);
    } else {
      column_types.emplace_back(col_type);
    }

    // Map each ORC column to its column
    orc_col_map[col] = column_types.size() - 1;
  }

  // If no rows or stripes to read, return empty columns
  if (num_rows <= 0 || selected_stripes.empty()) {
    std::transform(column_types.cbegin(),
                   column_types.cend(),
                   std::back_inserter(out_columns),
                   [](auto const &dtype) { return make_empty_column(dtype); });
  } else {
    // Get the total number of stripes across all input files.
    size_t total_num_stripes =
      std::accumulate(selected_stripes.begin(),
                      selected_stripes.end(),
                      0,
                      [](size_t sum, auto &stripe_source_mapping) {
                        return sum + stripe_source_mapping.stripe_info.size();
                      });

    const auto num_columns = _selected_columns.size();
    const auto num_chunks  = total_num_stripes * num_columns;
    hostdevice_vector<gpu::ColumnDesc> chunks(num_chunks, stream);
    memset(chunks.host_ptr(), 0, chunks.memory_size());

    const bool use_index =
      (_use_index == true) &&
      // Only use if we don't have much work with complete columns & stripes
      // TODO: Consider nrows, gpu, and tune the threshold
      (num_rows > _metadata->get_row_index_stride() && !(_metadata->get_row_index_stride() & 7) &&
       _metadata->get_row_index_stride() > 0 && num_columns * total_num_stripes < 8 * 128) &&
      // Only use if first row is aligned to a stripe boundary
      // TODO: Fix logic to handle unaligned rows
      (skip_rows == 0);

    // Logically view streams as columns
    std::vector<orc_stream_info> stream_info;

    // Tracker for eventually deallocating compressed and uncompressed data
    std::vector<rmm::device_buffer> stripe_data;

    size_t stripe_start_row   = 0;
    size_t num_dict_entries   = 0;
    size_t num_rowgroups      = 0;
    size_t stripe_chunk_index = 0;

    for (auto &stripe_source_mapping : selected_stripes) {
      // Iterate through the source files selected stripes
      for (size_t stripe_pos_index = 0; stripe_pos_index < stripe_source_mapping.stripe_info.size();
           stripe_pos_index++) {
        auto &stripe_pair        = stripe_source_mapping.stripe_info[stripe_pos_index];
        const auto stripe_info   = stripe_pair.first;
        const auto stripe_footer = stripe_pair.second;

        auto stream_count          = stream_info.size();
        const auto total_data_size = gather_stream_info(stripe_chunk_index,
                                                        stripe_info,
                                                        stripe_footer,
                                                        orc_col_map,
                                                        _selected_columns,
                                                        _metadata->get_types(),
                                                        use_index,
                                                        &num_dict_entries,
                                                        chunks,
                                                        stream_info);

        CUDF_EXPECTS(total_data_size > 0, "Expected streams data within stripe");

        stripe_data.emplace_back(total_data_size, stream);
        auto dst_base = static_cast<uint8_t *>(stripe_data.back().data());

        // Coalesce consecutive streams into one read
        while (stream_count < stream_info.size()) {
          const auto d_dst  = dst_base + stream_info[stream_count].dst_pos;
          const auto offset = stream_info[stream_count].offset;
          auto len          = stream_info[stream_count].length;
          stream_count++;

          while (stream_count < stream_info.size() &&
                 stream_info[stream_count].offset == offset + len) {
            len += stream_info[stream_count].length;
            stream_count++;
          }
          if (_metadata->per_file_metadata[stripe_source_mapping.source_idx]
                .source->is_device_read_preferred(len)) {
            CUDF_EXPECTS(
              _metadata->per_file_metadata[stripe_source_mapping.source_idx].source->device_read(
                offset, len, d_dst, stream) == len,
              "Unexpected discrepancy in bytes read.");
          } else {
            const auto buffer =
              _metadata->per_file_metadata[stripe_source_mapping.source_idx].source->host_read(
                offset, len);
            CUDF_EXPECTS(buffer->size() == len, "Unexpected discrepancy in bytes read.");
            CUDA_TRY(
              hipMemcpyAsync(d_dst, buffer->data(), len, hipMemcpyHostToDevice, stream.value()));
            stream.synchronize();
          }
        }

        // Update chunks to reference streams pointers
        for (size_t col_idx = 0; col_idx < num_columns; col_idx++) {
          auto &chunk         = chunks[stripe_chunk_index * num_columns + col_idx];
          chunk.start_row     = stripe_start_row;
          chunk.num_rows      = stripe_info->numberOfRows;
          chunk.encoding_kind = stripe_footer->columns[_selected_columns[col_idx]].kind;
          chunk.type_kind     = _metadata->per_file_metadata[stripe_source_mapping.source_idx]
                              .ff.types[_selected_columns[col_idx]]
                              .kind;
          auto const decimal_as_float64 = should_convert_decimal_column_to_float(
            _decimal_cols_as_float, _metadata->per_file_metadata[0], _selected_columns[col_idx]);
          chunk.decimal_scale = _metadata->per_file_metadata[stripe_source_mapping.source_idx]
                                  .ff.types[_selected_columns[col_idx]]
                                  .scale.value_or(0) |
                                (decimal_as_float64 ? orc::gpu::orc_decimal2float64_scale : 0);
          chunk.rowgroup_id = num_rowgroups;
          chunk.dtype_len   = (column_types[col_idx].id() == type_id::STRING)
                              ? sizeof(string_index_pair)
                              : cudf::size_of(column_types[col_idx]);
          if (chunk.type_kind == orc::TIMESTAMP) {
            chunk.ts_clock_rate = to_clockrate(_timestamp_type.id());
          }
          for (int k = 0; k < gpu::CI_NUM_STREAMS; k++) {
            chunk.streams[k] = dst_base + stream_info[chunk.strm_id[k]].dst_pos;
          }
        }

        stripe_start_row += stripe_info->numberOfRows;
        if (use_index) {
          num_rowgroups += (stripe_info->numberOfRows + _metadata->get_row_index_stride() - 1) /
                           _metadata->get_row_index_stride();
        }
        stripe_chunk_index++;
      }
    }

    // Process dataset chunk pages into output columns
    if (stripe_data.size() != 0) {
      // Setup row group descriptors if using indexes
      rmm::device_uvector<gpu::RowGroup> row_groups(num_rowgroups * num_columns, stream);
      if (_metadata->per_file_metadata[0].ps.compression != orc::NONE) {
        auto decomp_data =
          decompress_stripe_data(chunks,
                                 stripe_data,
                                 _metadata->per_file_metadata[0].decompressor.get(),
                                 stream_info,
                                 total_num_stripes,
                                 row_groups,
                                 _metadata->get_row_index_stride(),
                                 stream);
        stripe_data.clear();
        stripe_data.push_back(std::move(decomp_data));
      } else {
        if (not row_groups.is_empty()) {
          chunks.host_to_device(stream);
          gpu::ParseRowGroupIndex(row_groups.data(),
                                  nullptr,
                                  chunks.device_ptr(),
                                  num_columns,
                                  total_num_stripes,
                                  num_rowgroups,
                                  _metadata->get_row_index_stride(),
                                  stream);
        }
      }

      // Setup table for converting timestamp columns from local to UTC time
      auto const tz_table = _has_timestamp_column
                              ? build_timezone_transition_table(
                                  selected_stripes[0].stripe_info[0].second->writerTimezone, stream)
                              : timezone_table{};

      std::vector<column_buffer> out_buffers;
      for (size_t i = 0; i < column_types.size(); ++i) {
        bool is_nullable = false;
        for (size_t j = 0; j < total_num_stripes; ++j) {
          if (chunks[j * num_columns + i].strm_len[gpu::CI_PRESENT] != 0) {
            is_nullable = true;
            break;
          }
        }
        out_buffers.emplace_back(column_types[i], num_rows, is_nullable, stream, _mr);
      }

      decode_stream_data(chunks,
                         num_dict_entries,
                         skip_rows,
                         num_rows,
                         tz_table.view(),
                         row_groups,
                         _metadata->get_row_index_stride(),
                         out_buffers,
                         stream);

      for (size_t i = 0; i < column_types.size(); ++i) {
        out_columns.emplace_back(make_column(out_buffers[i], nullptr, stream, _mr));
      }
    }
  }

  // Return column names (must match order of returned columns)
  out_metadata.column_names.resize(_selected_columns.size());
  for (size_t i = 0; i < _selected_columns.size(); i++) {
    out_metadata.column_names[i] = _metadata->get_column_name(0, _selected_columns[i]);
  }

  for (const auto &meta : _metadata->per_file_metadata) {
    for (const auto &kv : meta.ff.metadata) { out_metadata.user_data.insert({kv.name, kv.value}); }
  }

  return {std::make_unique<table>(std::move(out_columns)), std::move(out_metadata)};
}

// Forward to implementation
reader::reader(std::vector<std::string> const &filepaths,
               orc_reader_options const &options,
               rmm::cuda_stream_view stream,
               rmm::mr::device_memory_resource *mr)
{
  _impl = std::make_unique<impl>(datasource::create(filepaths), options, mr);
}

// Forward to implementation
reader::reader(std::vector<std::unique_ptr<cudf::io::datasource>> &&sources,
               orc_reader_options const &options,
               rmm::cuda_stream_view stream,
               rmm::mr::device_memory_resource *mr)
{
  _impl = std::make_unique<impl>(std::move(sources), options, mr);
}

// Destructor within this translation unit
reader::~reader() = default;

// Forward to implementation
table_with_metadata reader::read(orc_reader_options const &options, rmm::cuda_stream_view stream)
{
  return _impl->read(
    options.get_skip_rows(), options.get_num_rows(), options.get_stripes(), stream);
}
}  // namespace orc
}  // namespace detail
}  // namespace io
}  // namespace cudf
