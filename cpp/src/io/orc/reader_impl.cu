#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019-2020, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

/**
 * @file reader_impl.cu
 * @brief cuDF-IO ORC reader class implementation
 */

#include "reader_impl.hpp"
#include "timezone.cuh"

#include <io/comp/gpuinflate.h>
#include <io/orc/orc.h>

#include <cudf/table/table.hpp>
#include <cudf/utilities/error.hpp>
#include <cudf/utilities/traits.hpp>

#include <rmm/cuda_stream_view.hpp>
#include <rmm/device_buffer.hpp>
#include <rmm/device_vector.hpp>

#include <algorithm>
#include <array>

namespace cudf {
namespace io {
namespace detail {
namespace orc {
// Import functionality that's independent of legacy code
using namespace cudf::io::orc;
using namespace cudf::io;

namespace {
/**
 * @brief Function that translates ORC data kind to cuDF type enum
 */
constexpr type_id to_type_id(const orc::SchemaType &schema,
                             bool use_np_dtypes,
                             type_id timestamp_type_id,
                             bool decimals_as_float64)
{
  switch (schema.kind) {
    case orc::BOOLEAN: return type_id::BOOL8;
    case orc::BYTE: return type_id::INT8;
    case orc::SHORT: return type_id::INT16;
    case orc::INT: return type_id::INT32;
    case orc::LONG: return type_id::INT64;
    case orc::FLOAT: return type_id::FLOAT32;
    case orc::DOUBLE: return type_id::FLOAT64;
    case orc::STRING:
    case orc::BINARY:
    case orc::VARCHAR:
    case orc::CHAR:
      // Variable-length types can all be mapped to STRING
      return type_id::STRING;
    case orc::TIMESTAMP:
      return (timestamp_type_id != type_id::EMPTY) ? timestamp_type_id
                                                   : type_id::TIMESTAMP_NANOSECONDS;
    case orc::DATE:
      // There isn't a (DAYS -> np.dtype) mapping
      return (use_np_dtypes) ? type_id::TIMESTAMP_MILLISECONDS : type_id::TIMESTAMP_DAYS;
    case orc::DECIMAL:
      // There isn't an arbitrary-precision type in cuDF, so map as float or int
      return (decimals_as_float64) ? type_id::FLOAT64 : type_id::INT64;
    default: break;
  }

  return type_id::EMPTY;
}

/**
 * @brief Function that translates cuDF time unit to ORC clock frequency
 */
constexpr int32_t to_clockrate(type_id timestamp_type_id)
{
  switch (timestamp_type_id) {
    case type_id::TIMESTAMP_SECONDS: return 1;
    case type_id::TIMESTAMP_MILLISECONDS: return 1000;
    case type_id::TIMESTAMP_MICROSECONDS: return 1000000;
    case type_id::TIMESTAMP_NANOSECONDS: return 1000000000;
    default: return 0;
  }
}

constexpr std::pair<gpu::StreamIndexType, uint32_t> get_index_type_and_pos(
  const orc::StreamKind kind, uint32_t skip_count, bool non_child)
{
  switch (kind) {
    case orc::DATA:
      skip_count += 1;
      skip_count |= (skip_count & 0xff) << 8;
      return std::make_pair(gpu::CI_DATA, skip_count);
    case orc::LENGTH:
    case orc::SECONDARY:
      skip_count += 1;
      skip_count |= (skip_count & 0xff) << 16;
      return std::make_pair(gpu::CI_DATA2, skip_count);
    case orc::DICTIONARY_DATA: return std::make_pair(gpu::CI_DICTIONARY, skip_count);
    case orc::PRESENT:
      skip_count += (non_child ? 1 : 0);
      return std::make_pair(gpu::CI_PRESENT, skip_count);
    case orc::ROW_INDEX: return std::make_pair(gpu::CI_INDEX, skip_count);
    default:
      // Skip this stream as it's not strictly required
      return std::make_pair(gpu::CI_NUM_STREAMS, 0);
  }
}

}  // namespace

namespace {
/**
 * @brief Struct that maps ORC streams to columns
 */
struct orc_stream_info {
  orc_stream_info() = default;
  explicit orc_stream_info(
    uint64_t offset_, size_t dst_pos_, uint32_t length_, uint32_t gdf_idx_, uint32_t stripe_idx_)
    : offset(offset_),
      dst_pos(dst_pos_),
      length(length_),
      gdf_idx(gdf_idx_),
      stripe_idx(stripe_idx_)
  {
  }
  uint64_t offset;      // offset in file
  size_t dst_pos;       // offset in memory relative to start of compressed stripe data
  uint32_t length;      // length in file
  uint32_t gdf_idx;     // column index
  uint32_t stripe_idx;  // stripe index
};

/**
 * @brief Function that populates column descriptors stream/chunk
 */
size_t gather_stream_info(const size_t stripe_index,
                          const orc::StripeInformation *stripeinfo,
                          const orc::StripeFooter *stripefooter,
                          const std::vector<int> &orc2gdf,
                          const std::vector<int> &gdf2orc,
                          const std::vector<orc::SchemaType> types,
                          bool use_index,
                          size_t *num_dictionary_entries,
                          hostdevice_vector<gpu::ColumnDesc> &chunks,
                          std::vector<orc_stream_info> &stream_info)
{
  const auto num_columns = gdf2orc.size();
  uint64_t src_offset    = 0;
  uint64_t dst_offset    = 0;
  for (const auto &stream : stripefooter->streams) {
    if (stream.column >= orc2gdf.size()) {
      dst_offset += stream.length;
      continue;
    }

    auto col = orc2gdf[stream.column];
    if (col == -1) {
      // A struct-type column has no data itself, but rather child columns
      // for each of its fields. There is only a PRESENT stream, which
      // needs to be included for the reader.
      const auto schema_type = types[stream.column];
      if (schema_type.subtypes.size() != 0) {
        if (schema_type.kind == orc::STRUCT && stream.kind == orc::PRESENT) {
          for (const auto &idx : schema_type.subtypes) {
            auto child_idx = (idx < orc2gdf.size()) ? orc2gdf[idx] : -1;
            if (child_idx >= 0) {
              col                             = child_idx;
              auto &chunk                     = chunks[stripe_index * num_columns + col];
              chunk.strm_id[gpu::CI_PRESENT]  = stream_info.size();
              chunk.strm_len[gpu::CI_PRESENT] = stream.length;
            }
          }
        }
      }
    }
    if (col != -1) {
      if (src_offset >= stripeinfo->indexLength || use_index) {
        // NOTE: skip_count field is temporarily used to track index ordering
        auto &chunk = chunks[stripe_index * num_columns + col];
        const auto idx =
          get_index_type_and_pos(stream.kind, chunk.skip_count, col == orc2gdf[stream.column]);
        if (idx.first < gpu::CI_NUM_STREAMS) {
          chunk.strm_id[idx.first]  = stream_info.size();
          chunk.strm_len[idx.first] = stream.length;
          chunk.skip_count          = idx.second;

          if (idx.first == gpu::CI_DICTIONARY) {
            chunk.dictionary_start = *num_dictionary_entries;
            chunk.dict_len         = stripefooter->columns[stream.column].dictionarySize;
            *num_dictionary_entries += stripefooter->columns[stream.column].dictionarySize;
          }
        }
      }
      stream_info.emplace_back(
        stripeinfo->offset + src_offset, dst_offset, stream.length, col, stripe_index);
      dst_offset += stream.length;
    }
    src_offset += stream.length;
  }

  return dst_offset;
}

}  // namespace

rmm::device_buffer reader::impl::decompress_stripe_data(
  hostdevice_vector<gpu::ColumnDesc> &chunks,
  const std::vector<rmm::device_buffer> &stripe_data,
  const OrcDecompressor *decompressor,
  std::vector<orc_stream_info> &stream_info,
  size_t num_stripes,
  rmm::device_vector<gpu::RowGroup> &row_groups,
  size_t row_index_stride,
  rmm::cuda_stream_view stream)
{
  // Parse the columns' compressed info
  hostdevice_vector<gpu::CompressedStreamInfo> compinfo(0, stream_info.size(), stream);
  for (const auto &info : stream_info) {
    compinfo.insert(gpu::CompressedStreamInfo(
      static_cast<const uint8_t *>(stripe_data[info.stripe_idx].data()) + info.dst_pos,
      info.length));
  }
  CUDA_TRY(hipMemcpyAsync(compinfo.device_ptr(),
                           compinfo.host_ptr(),
                           compinfo.memory_size(),
                           hipMemcpyHostToDevice,
                           stream.value()));
  gpu::ParseCompressedStripeData(compinfo.device_ptr(),
                                 compinfo.size(),
                                 decompressor->GetBlockSize(),
                                 decompressor->GetLog2MaxCompressionRatio(),
                                 stream);
  CUDA_TRY(hipMemcpyAsync(compinfo.host_ptr(),
                           compinfo.device_ptr(),
                           compinfo.memory_size(),
                           hipMemcpyDeviceToHost,
                           stream.value()));
  stream.synchronize();

  // Count the exact number of compressed blocks
  size_t num_compressed_blocks   = 0;
  size_t num_uncompressed_blocks = 0;
  size_t total_decomp_size       = 0;
  for (size_t i = 0; i < compinfo.size(); ++i) {
    num_compressed_blocks += compinfo[i].num_compressed_blocks;
    num_uncompressed_blocks += compinfo[i].num_uncompressed_blocks;
    total_decomp_size += compinfo[i].max_uncompressed_size;
  }
  CUDF_EXPECTS(total_decomp_size > 0, "No decompressible data found");

  rmm::device_buffer decomp_data(total_decomp_size, stream);
  rmm::device_vector<gpu_inflate_input_s> inflate_in(num_compressed_blocks +
                                                     num_uncompressed_blocks);
  rmm::device_vector<gpu_inflate_status_s> inflate_out(num_compressed_blocks);

  // Parse again to populate the decompression input/output buffers
  size_t decomp_offset      = 0;
  uint32_t start_pos        = 0;
  uint32_t start_pos_uncomp = (uint32_t)num_compressed_blocks;
  for (size_t i = 0; i < compinfo.size(); ++i) {
    auto dst_base                 = static_cast<uint8_t *>(decomp_data.data());
    compinfo[i].uncompressed_data = dst_base + decomp_offset;
    compinfo[i].decctl            = inflate_in.data().get() + start_pos;
    compinfo[i].decstatus         = inflate_out.data().get() + start_pos;
    compinfo[i].copyctl           = inflate_in.data().get() + start_pos_uncomp;

    stream_info[i].dst_pos = decomp_offset;
    decomp_offset += compinfo[i].max_uncompressed_size;
    start_pos += compinfo[i].num_compressed_blocks;
    start_pos_uncomp += compinfo[i].num_uncompressed_blocks;
  }
  CUDA_TRY(hipMemcpyAsync(compinfo.device_ptr(),
                           compinfo.host_ptr(),
                           compinfo.memory_size(),
                           hipMemcpyHostToDevice,
                           stream.value()));
  gpu::ParseCompressedStripeData(compinfo.device_ptr(),
                                 compinfo.size(),
                                 decompressor->GetBlockSize(),
                                 decompressor->GetLog2MaxCompressionRatio(),
                                 stream);

  // Dispatch batches of blocks to decompress
  if (num_compressed_blocks > 0) {
    switch (decompressor->GetKind()) {
      case orc::ZLIB:
        CUDA_TRY(gpuinflate(
          inflate_in.data().get(), inflate_out.data().get(), num_compressed_blocks, 0, stream));
        break;
      case orc::SNAPPY:
        CUDA_TRY(gpu_unsnap(
          inflate_in.data().get(), inflate_out.data().get(), num_compressed_blocks, stream));
        break;
      default: CUDF_EXPECTS(false, "Unexpected decompression dispatch"); break;
    }
  }
  if (num_uncompressed_blocks > 0) {
    CUDA_TRY(gpu_copy_uncompressed_blocks(
      inflate_in.data().get() + num_compressed_blocks, num_uncompressed_blocks, stream));
  }
  gpu::PostDecompressionReassemble(compinfo.device_ptr(), compinfo.size(), stream);

  // Update the stream information with the updated uncompressed info
  // TBD: We could update the value from the information we already
  // have in stream_info[], but using the gpu results also updates
  // max_uncompressed_size to the actual uncompressed size, or zero if
  // decompression failed.
  CUDA_TRY(hipMemcpyAsync(compinfo.host_ptr(),
                           compinfo.device_ptr(),
                           compinfo.memory_size(),
                           hipMemcpyDeviceToHost,
                           stream.value()));
  stream.synchronize();

  const size_t num_columns = chunks.size() / num_stripes;

  for (size_t i = 0; i < num_stripes; ++i) {
    for (size_t j = 0; j < num_columns; ++j) {
      auto &chunk = chunks[i * num_columns + j];
      for (int k = 0; k < gpu::CI_NUM_STREAMS; ++k) {
        if (chunk.strm_len[k] > 0 && chunk.strm_id[k] < compinfo.size()) {
          chunk.streams[k]  = compinfo[chunk.strm_id[k]].uncompressed_data;
          chunk.strm_len[k] = compinfo[chunk.strm_id[k]].max_uncompressed_size;
        }
      }
    }
  }

  if (not row_groups.empty()) {
    CUDA_TRY(hipMemcpyAsync(chunks.device_ptr(),
                             chunks.host_ptr(),
                             chunks.memory_size(),
                             hipMemcpyHostToDevice,
                             stream.value()));
    gpu::ParseRowGroupIndex(row_groups.data().get(),
                            compinfo.device_ptr(),
                            chunks.device_ptr(),
                            num_columns,
                            num_stripes,
                            row_groups.size() / num_columns,
                            row_index_stride,
                            stream);
  }

  return decomp_data;
}

void reader::impl::decode_stream_data(hostdevice_vector<gpu::ColumnDesc> &chunks,
                                      size_t num_dicts,
                                      size_t skip_rows,
                                      size_t num_rows,
                                      timezone_table const &tz_table,
                                      const rmm::device_vector<gpu::RowGroup> &row_groups,
                                      size_t row_index_stride,
                                      std::vector<column_buffer> &out_buffers,
                                      rmm::cuda_stream_view stream)
{
  const auto num_columns = out_buffers.size();
  const auto num_stripes = chunks.size() / out_buffers.size();

  // Update chunks with pointers to column data
  for (size_t i = 0; i < num_stripes; ++i) {
    for (size_t j = 0; j < num_columns; ++j) {
      auto &chunk            = chunks[i * num_columns + j];
      chunk.column_data_base = out_buffers[j].data();
      chunk.valid_map_base   = out_buffers[j].null_mask();
    }
  }

  // Allocate global dictionary for deserializing
  rmm::device_vector<gpu::DictionaryEntry> global_dict(num_dicts);

  CUDA_TRY(hipMemcpyAsync(chunks.device_ptr(),
                           chunks.host_ptr(),
                           chunks.memory_size(),
                           hipMemcpyHostToDevice,
                           stream.value()));
  gpu::DecodeNullsAndStringDictionaries(chunks.device_ptr(),
                                        global_dict.data().get(),
                                        num_columns,
                                        num_stripes,
                                        num_rows,
                                        skip_rows,
                                        stream);
  gpu::DecodeOrcColumnData(chunks.device_ptr(),
                           global_dict.data().get(),
                           num_columns,
                           num_stripes,
                           num_rows,
                           skip_rows,
                           tz_table.view(),
                           row_groups.data().get(),
                           row_groups.size() / num_columns,
                           row_index_stride,
                           stream);
  CUDA_TRY(hipMemcpyAsync(chunks.host_ptr(),
                           chunks.device_ptr(),
                           chunks.memory_size(),
                           hipMemcpyDeviceToHost,
                           stream.value()));
  stream.synchronize();

  for (size_t i = 0; i < num_stripes; ++i) {
    for (size_t j = 0; j < num_columns; ++j) {
      out_buffers[j].null_count() += chunks[i * num_columns + j].null_count;
    }
  }
}

reader::impl::impl(std::unique_ptr<datasource> source,
                   orc_reader_options const &options,
                   rmm::mr::device_memory_resource *mr)
  : _mr(mr), _source(std::move(source))
{
  // Open and parse the source dataset metadata
  _metadata = std::make_unique<cudf::io::orc::metadata>(_source.get());

  // Select only columns required by the options
  _selected_columns = _metadata->select_columns(options.get_columns(), _has_timestamp_column);

  // Override output timestamp resolution if requested
  if (options.get_timestamp_type().id() != type_id::EMPTY) {
    _timestamp_type = options.get_timestamp_type();
  }

  // Enable or disable attempt to use row index for parsing
  _use_index = options.is_enabled_use_index();

  // Enable or disable the conversion to numpy-compatible dtypes
  _use_np_dtypes = options.is_enabled_use_np_dtypes();

  // Control decimals conversion (float64 or int64 with optional scale)
  _decimals_as_float64   = options.is_enabled_decimals_as_float64();
  _decimals_as_int_scale = options.get_forced_decimals_scale();
}

table_with_metadata reader::impl::read(size_type skip_rows,
                                       size_type num_rows,
                                       const std::vector<size_type> &stripes,
                                       rmm::cuda_stream_view stream)
{
  std::vector<std::unique_ptr<column>> out_columns;
  table_metadata out_metadata;

  // Select only stripes required (aka row groups)
  const auto selected_stripes = _metadata->select_stripes(stripes, skip_rows, num_rows);

  // Association between each ORC column and its cudf::column
  std::vector<int32_t> orc_col_map(_metadata->get_num_columns(), -1);

  // Get a list of column data types
  std::vector<data_type> column_types;
  for (const auto &col : _selected_columns) {
    auto col_type = to_type_id(
      _metadata->ff.types[col], _use_np_dtypes, _timestamp_type.id(), _decimals_as_float64);
    CUDF_EXPECTS(col_type != type_id::EMPTY, "Unknown type");
    column_types.emplace_back(col_type);

    // Map each ORC column to its column
    orc_col_map[col] = column_types.size() - 1;
  }

  // If no rows or stripes to read, return empty columns
  if (num_rows <= 0 || selected_stripes.empty()) {
    std::transform(column_types.cbegin(),
                   column_types.cend(),
                   std::back_inserter(out_columns),
                   [](auto const &dtype) { return make_empty_column(dtype); });
  } else {
    const auto num_columns = _selected_columns.size();
    const auto num_chunks  = selected_stripes.size() * num_columns;
    hostdevice_vector<gpu::ColumnDesc> chunks(num_chunks, stream);
    memset(chunks.host_ptr(), 0, chunks.memory_size());

    const bool use_index =
      (_use_index == true) &&
      // Only use if we don't have much work with complete columns & stripes
      // TODO: Consider nrows, gpu, and tune the threshold
      (num_rows > _metadata->get_row_index_stride() && !(_metadata->get_row_index_stride() & 7) &&
       _metadata->get_row_index_stride() > 0 && num_columns * selected_stripes.size() < 8 * 128) &&
      // Only use if first row is aligned to a stripe boundary
      // TODO: Fix logic to handle unaligned rows
      (skip_rows == 0);

    // Logically view streams as columns
    std::vector<orc_stream_info> stream_info;

    // Tracker for eventually deallocating compressed and uncompressed data
    std::vector<rmm::device_buffer> stripe_data;

    size_t stripe_start_row = 0;
    size_t num_dict_entries = 0;
    size_t num_rowgroups    = 0;
    for (size_t i = 0; i < selected_stripes.size(); ++i) {
      const auto stripe_info   = selected_stripes[i].first;
      const auto stripe_footer = selected_stripes[i].second;

      auto stream_count          = stream_info.size();
      const auto total_data_size = gather_stream_info(i,
                                                      stripe_info,
                                                      stripe_footer,
                                                      orc_col_map,
                                                      _selected_columns,
                                                      _metadata->ff.types,
                                                      use_index,
                                                      &num_dict_entries,
                                                      chunks,
                                                      stream_info);
      CUDF_EXPECTS(total_data_size > 0, "Expected streams data within stripe");

      stripe_data.emplace_back(total_data_size, stream);
      auto dst_base = static_cast<uint8_t *>(stripe_data.back().data());

      // Coalesce consecutive streams into one read
      while (stream_count < stream_info.size()) {
        const auto d_dst  = dst_base + stream_info[stream_count].dst_pos;
        const auto offset = stream_info[stream_count].offset;
        auto len          = stream_info[stream_count].length;
        stream_count++;

        while (stream_count < stream_info.size() &&
               stream_info[stream_count].offset == offset + len) {
          len += stream_info[stream_count].length;
          stream_count++;
        }
        const auto buffer = _source->host_read(offset, len);
        CUDA_TRY(
          hipMemcpyAsync(d_dst, buffer->data(), len, hipMemcpyHostToDevice, stream.value()));
        stream.synchronize();
      }

      // Update chunks to reference streams pointers
      for (size_t j = 0; j < num_columns; j++) {
        auto &chunk         = chunks[i * num_columns + j];
        chunk.start_row     = stripe_start_row;
        chunk.num_rows      = stripe_info->numberOfRows;
        chunk.encoding_kind = stripe_footer->columns[_selected_columns[j]].kind;
        chunk.type_kind     = _metadata->ff.types[_selected_columns[j]].kind;
        if (_decimals_as_float64) {
          chunk.decimal_scale =
            _metadata->ff.types[_selected_columns[j]].scale | orc::gpu::orc_decimal2float64_scale;
        } else if (_decimals_as_int_scale < 0) {
          chunk.decimal_scale = _metadata->ff.types[_selected_columns[j]].scale;
        } else {
          chunk.decimal_scale = _decimals_as_int_scale;
        }
        chunk.rowgroup_id = num_rowgroups;
        chunk.dtype_len   = (column_types[j].id() == type_id::STRING)
                            ? sizeof(std::pair<const char *, size_t>)
                            : cudf::size_of(column_types[j]);
        if (chunk.type_kind == orc::TIMESTAMP) {
          chunk.ts_clock_rate = to_clockrate(_timestamp_type.id());
        }
        for (int k = 0; k < gpu::CI_NUM_STREAMS; k++) {
          if (chunk.strm_len[k] > 0) {
            chunk.streams[k] = dst_base + stream_info[chunk.strm_id[k]].dst_pos;
          }
        }
      }
      stripe_start_row += stripe_info->numberOfRows;
      if (use_index) {
        num_rowgroups += (stripe_info->numberOfRows + _metadata->get_row_index_stride() - 1) /
                         _metadata->get_row_index_stride();
      }
    }

    // Process dataset chunk pages into output columns
    if (stripe_data.size() != 0) {
      // Setup row group descriptors if using indexes
      rmm::device_vector<gpu::RowGroup> row_groups(num_rowgroups * num_columns);
      if (_metadata->ps.compression != orc::NONE) {
        auto decomp_data = decompress_stripe_data(chunks,
                                                  stripe_data,
                                                  _metadata->decompressor.get(),
                                                  stream_info,
                                                  selected_stripes.size(),
                                                  row_groups,
                                                  _metadata->get_row_index_stride(),
                                                  stream);
        stripe_data.clear();
        stripe_data.push_back(std::move(decomp_data));
      } else {
        if (not row_groups.empty()) {
          CUDA_TRY(hipMemcpyAsync(chunks.device_ptr(),
                                   chunks.host_ptr(),
                                   chunks.memory_size(),
                                   hipMemcpyHostToDevice,
                                   stream.value()));
          gpu::ParseRowGroupIndex(row_groups.data().get(),
                                  nullptr,
                                  chunks.device_ptr(),
                                  num_columns,
                                  selected_stripes.size(),
                                  num_rowgroups,
                                  _metadata->get_row_index_stride(),
                                  stream);
        }
      }

      // Setup table for converting timestamp columns from local to UTC time
      auto const tz_table =
        _has_timestamp_column
          ? build_timezone_transition_table(selected_stripes[0].second->writerTimezone)
          : timezone_table{};

      std::vector<column_buffer> out_buffers;
      for (size_t i = 0; i < column_types.size(); ++i) {
        bool is_nullable = false;
        for (size_t j = 0; j < selected_stripes.size(); ++j) {
          if (chunks[j * num_columns + i].strm_len[gpu::CI_PRESENT] != 0) {
            is_nullable = true;
            break;
          }
        }
        out_buffers.emplace_back(column_types[i], num_rows, is_nullable, stream, _mr);
      }

      decode_stream_data(chunks,
                         num_dict_entries,
                         skip_rows,
                         num_rows,
                         tz_table,
                         row_groups,
                         _metadata->get_row_index_stride(),
                         out_buffers,
                         stream);

      for (size_t i = 0; i < column_types.size(); ++i) {
        out_columns.emplace_back(make_column(out_buffers[i], nullptr, stream, _mr));
      }
    }
  }

  // Return column names (must match order of returned columns)
  out_metadata.column_names.resize(_selected_columns.size());
  for (size_t i = 0; i < _selected_columns.size(); i++) {
    out_metadata.column_names[i] = _metadata->get_column_name(_selected_columns[i]);
  }
  // Return user metadata
  for (const auto &kv : _metadata->ff.metadata) {
    out_metadata.user_data.insert({kv.name, kv.value});
  }

  return {std::make_unique<table>(std::move(out_columns)), std::move(out_metadata)};
}

// Forward to implementation
reader::reader(std::vector<std::string> const &filepaths,
               orc_reader_options const &options,
               rmm::mr::device_memory_resource *mr)
{
  CUDF_EXPECTS(filepaths.size() == 1, "Only a single source is currently supported.");
  _impl = std::make_unique<impl>(datasource::create(filepaths[0]), options, mr);
}

// Forward to implementation
reader::reader(std::vector<std::unique_ptr<cudf::io::datasource>> &&sources,
               orc_reader_options const &options,
               rmm::mr::device_memory_resource *mr)
{
  CUDF_EXPECTS(sources.size() == 1, "Only a single source is currently supported.");
  _impl = std::make_unique<impl>(std::move(sources[0]), options, mr);
}

// Destructor within this translation unit
reader::~reader() = default;

// Forward to implementation
table_with_metadata reader::read(orc_reader_options const &options, rmm::cuda_stream_view stream)
{
  return _impl->read(
    options.get_skip_rows(), options.get_num_rows(), options.get_stripes(), stream);
}
}  // namespace orc
}  // namespace detail
}  // namespace io
}  // namespace cudf
