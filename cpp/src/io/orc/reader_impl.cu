#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019-2021, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

/**
 * @file reader_impl.cu
 * @brief cuDF-IO ORC reader class implementation
 */

#include "io/orc/orc_gpu.h"
#include "reader_impl.hpp"
#include "timezone.cuh"

#include <io/comp/gpuinflate.h>
#include "orc.h"

#include <cudf/detail/utilities/vector_factories.hpp>
#include <cudf/table/table.hpp>
#include <cudf/utilities/error.hpp>
#include <cudf/utilities/traits.hpp>

#include <rmm/cuda_stream_view.hpp>
#include <rmm/device_buffer.hpp>
#include <rmm/device_uvector.hpp>
#include <rmm/exec_policy.hpp>

#include <algorithm>
#include <array>

namespace cudf {
namespace io {
namespace detail {
namespace orc {
// Import functionality that's independent of legacy code
using namespace cudf::io::orc;
using namespace cudf::io;

namespace {
/**
 * @brief Function that translates ORC data kind to cuDF type enum
 */
constexpr type_id to_type_id(const orc::SchemaType &schema,
                             bool use_np_dtypes,
                             type_id timestamp_type_id)
{
  switch (schema.kind) {
    case orc::BOOLEAN: return type_id::BOOL8;
    case orc::BYTE: return type_id::INT8;
    case orc::SHORT: return type_id::INT16;
    case orc::INT: return type_id::INT32;
    case orc::LONG: return type_id::INT64;
    case orc::FLOAT: return type_id::FLOAT32;
    case orc::DOUBLE: return type_id::FLOAT64;
    case orc::STRING:
    case orc::BINARY:
    case orc::VARCHAR:
    case orc::CHAR:
      // Variable-length types can all be mapped to STRING
      return type_id::STRING;
    case orc::TIMESTAMP:
      return (timestamp_type_id != type_id::EMPTY) ? timestamp_type_id
                                                   : type_id::TIMESTAMP_NANOSECONDS;
    case orc::DATE:
      // There isn't a (DAYS -> np.dtype) mapping
      return (use_np_dtypes) ? type_id::TIMESTAMP_MILLISECONDS : type_id::TIMESTAMP_DAYS;
    case orc::DECIMAL: return type_id::DECIMAL64;
    case orc::LIST: return type_id::LIST;
    case orc::STRUCT: return type_id::STRUCT;
    default: break;
  }

  return type_id::EMPTY;
}

/**
 * @brief Function that translates cuDF time unit to ORC clock frequency
 */
constexpr int32_t to_clockrate(type_id timestamp_type_id)
{
  switch (timestamp_type_id) {
    case type_id::TIMESTAMP_SECONDS: return 1;
    case type_id::TIMESTAMP_MILLISECONDS: return 1000;
    case type_id::TIMESTAMP_MICROSECONDS: return 1000000;
    case type_id::TIMESTAMP_NANOSECONDS: return 1000000000;
    default: return 0;
  }
}

constexpr std::pair<gpu::StreamIndexType, uint32_t> get_index_type_and_pos(
  const orc::StreamKind kind, uint32_t skip_count, bool non_child)
{
  switch (kind) {
    case orc::DATA:
      skip_count += 1;
      skip_count |= (skip_count & 0xff) << 8;
      return std::make_pair(gpu::CI_DATA, skip_count);
    case orc::LENGTH:
    case orc::SECONDARY:
      skip_count += 1;
      skip_count |= (skip_count & 0xff) << 16;
      return std::make_pair(gpu::CI_DATA2, skip_count);
    case orc::DICTIONARY_DATA: return std::make_pair(gpu::CI_DICTIONARY, skip_count);
    case orc::PRESENT:
      skip_count += (non_child ? 1 : 0);
      return std::make_pair(gpu::CI_PRESENT, skip_count);
    case orc::ROW_INDEX: return std::make_pair(gpu::CI_INDEX, skip_count);
    default:
      // Skip this stream as it's not strictly required
      return std::make_pair(gpu::CI_NUM_STREAMS, 0);
  }
}

}  // namespace

namespace {
/**
 * @brief Struct that maps ORC streams to columns
 */
struct orc_stream_info {
  orc_stream_info() = default;
  explicit orc_stream_info(
    uint64_t offset_, size_t dst_pos_, uint32_t length_, uint32_t gdf_idx_, uint32_t stripe_idx_)
    : offset(offset_),
      dst_pos(dst_pos_),
      length(length_),
      gdf_idx(gdf_idx_),
      stripe_idx(stripe_idx_)
  {
  }
  uint64_t offset;      // offset in file
  size_t dst_pos;       // offset in memory relative to start of compressed stripe data
  size_t length;        // length in file
  uint32_t gdf_idx;     // column index
  uint32_t stripe_idx;  // stripe index
};

/**
 * @brief Function that populates column descriptors stream/chunk
 */
size_t gather_stream_info(const size_t stripe_index,
                          const orc::StripeInformation *stripeinfo,
                          const orc::StripeFooter *stripefooter,
                          const std::vector<int> &orc2gdf,
                          const std::vector<orc_column_meta> &gdf2orc,
                          const std::vector<orc::SchemaType> types,
                          bool use_index,
                          size_t *num_dictionary_entries,
                          hostdevice_vector<gpu::ColumnDesc> &chunks,
                          std::vector<orc_stream_info> &stream_info)
{
  const auto num_columns = gdf2orc.size();
  uint64_t src_offset    = 0;
  uint64_t dst_offset    = 0;
  for (const auto &stream : stripefooter->streams) {
    if (!stream.column_id || *stream.column_id >= orc2gdf.size()) {
      dst_offset += stream.length;
      continue;
    }

    auto const column_id = *stream.column_id;
    auto col             = orc2gdf[column_id];

    if (col == -1) {
      // A struct-type column has no data itself, but rather child columns
      // for each of its fields. There is only a PRESENT stream, which
      // needs to be included for the reader.
      const auto schema_type = types[column_id];
      if (schema_type.subtypes.size() != 0) {
        if (schema_type.kind == orc::STRUCT && stream.kind == orc::PRESENT) {
          for (const auto &idx : schema_type.subtypes) {
            auto child_idx = (idx < orc2gdf.size()) ? orc2gdf[idx] : -1;
            if (child_idx >= 0) {
              col                             = child_idx;
              auto &chunk                     = chunks[stripe_index * num_columns + col];
              chunk.strm_id[gpu::CI_PRESENT]  = stream_info.size();
              chunk.strm_len[gpu::CI_PRESENT] = stream.length;
            }
          }
        }
      }
    }
    if (col != -1) {
      if (src_offset >= stripeinfo->indexLength || use_index) {
        // NOTE: skip_count field is temporarily used to track index ordering
        auto &chunk = chunks[stripe_index * num_columns + col];
        const auto idx =
          get_index_type_and_pos(stream.kind, chunk.skip_count, col == orc2gdf[column_id]);
        if (idx.first < gpu::CI_NUM_STREAMS) {
          chunk.strm_id[idx.first]  = stream_info.size();
          chunk.strm_len[idx.first] = stream.length;
          chunk.skip_count          = idx.second;

          if (idx.first == gpu::CI_DICTIONARY) {
            chunk.dictionary_start = *num_dictionary_entries;
            chunk.dict_len         = stripefooter->columns[column_id].dictionarySize;
            *num_dictionary_entries += stripefooter->columns[column_id].dictionarySize;
          }
        }
      }
      stream_info.emplace_back(
        stripeinfo->offset + src_offset, dst_offset, stream.length, col, stripe_index);
      dst_offset += stream.length;
    }
    src_offset += stream.length;
  }

  return dst_offset;
}

}  // namespace

/**
 * @brief In order to support multiple input files/buffers we need to gather
 * the metadata across all of those input(s). This class provides a place
 * to aggregate that metadata from all the files.
 */
class aggregate_orc_metadata {
  using OrcStripeInfo = std::pair<const StripeInformation *, const StripeFooter *>;

 public:
  mutable std::vector<cudf::io::orc::metadata> per_file_metadata;
  size_type const num_rows;
  size_type const num_columns;
  size_type const num_stripes;

  /**
   * @brief Create a metadata object from each element in the source vector
   */
  auto metadatas_from_sources(std::vector<std::unique_ptr<datasource>> const &sources)
  {
    std::vector<cudf::io::orc::metadata> metadatas;
    std::transform(
      sources.cbegin(), sources.cend(), std::back_inserter(metadatas), [](auto const &source) {
        return cudf::io::orc::metadata(source.get());
      });
    return metadatas;
  }

  /**
   * @brief Sums up the number of rows of each source
   */
  size_type calc_num_rows() const
  {
    return std::accumulate(
      per_file_metadata.begin(), per_file_metadata.end(), 0, [](auto &sum, auto &pfm) {
        return sum + pfm.get_total_rows();
      });
  }

  /**
   * @brief Number of columns in a ORC file.
   */
  size_type calc_num_cols() const
  {
    if (not per_file_metadata.empty()) { return per_file_metadata[0].get_num_columns(); }
    return 0;
  }

  /**
   * @brief Sums up the number of stripes of each source
   */
  size_type calc_num_stripes() const
  {
    return std::accumulate(
      per_file_metadata.begin(), per_file_metadata.end(), 0, [](auto &sum, auto &pfm) {
        return sum + pfm.get_num_stripes();
      });
  }

 public:
  aggregate_orc_metadata(std::vector<std::unique_ptr<datasource>> const &sources)
    : per_file_metadata(metadatas_from_sources(sources)),
      num_rows(calc_num_rows()),
      num_columns(calc_num_cols()),
      num_stripes(calc_num_stripes())
  {
    // Verify that the input files have matching numbers of columns
    int num_cols = -1;
    for (auto const &pfm : per_file_metadata) {
      if (num_cols == -1) { num_cols = pfm.get_num_columns(); }
      if (pfm.get_num_columns() != num_cols) {
        CUDF_EXPECTS(num_cols == static_cast<int>(pfm.get_num_columns()),
                     "All sources must have the same number of columns");
      }
    }

    // XXX: Need to talk with Vukasin about the best way to compare this schema ....
    // Comparing types is likely the best thing to do here.
    // // Verify that the input files have matching schemas
    // for (auto const &pfm : per_file_metadata) {
    //   CUDF_EXPECTS(per_file_metadata[0].schema == pfm.schema,
    //                "All sources must have the same schemas");
    // }
  }

  auto const &get_schema(int schema_idx) const { return per_file_metadata[0].ff.types[schema_idx]; }

  auto get_metadata_at_idx(int metadata_idx) const { return &per_file_metadata[metadata_idx]; };

  auto get_col_type(int col_idx) const { return per_file_metadata[0].ff.types[col_idx]; }

  size_type get_num_rows() const { return num_rows; }

  size_type get_num_cols() const { return per_file_metadata[0].get_num_columns(); }

  auto get_num_stripes() const { return num_stripes; }

  auto get_num_source_files() const { return per_file_metadata.size(); }

  auto get_types() const { return per_file_metadata[0].ff.types; }

  int get_row_index_stride() const { return per_file_metadata[0].ff.rowIndexStride; }

  auto get_column_name(const int source_idx, const int column_idx) const
  {
    return per_file_metadata[source_idx].get_column_name(column_idx);
  }

  std::vector<cudf::io::orc::metadata::stripe_source_mapping> select_stripes(
    std::vector<std::vector<size_type>> const &user_specified_stripes,
    size_type &row_start,
    size_type &row_count)
  {
    std::vector<cudf::io::orc::metadata::stripe_source_mapping> selected_stripes_mapping;

    if (!user_specified_stripes.empty()) {
      CUDF_EXPECTS(user_specified_stripes.size() == get_num_source_files(),
                   "Must specify stripes for each source");
      // row_start is 0 if stripes are set. If this is not true anymore, then
      // row_start needs to be subtracted to get the correct row_count
      CUDF_EXPECTS(row_start == 0, "Start row index should be 0");

      row_count = 0;
      // Each vector entry represents a source file; each nested vector represents the
      // user_defined_stripes to get from that source file
      for (size_t src_file_idx = 0; src_file_idx < user_specified_stripes.size(); ++src_file_idx) {
        std::vector<int> stripe_idxs;
        std::vector<OrcStripeInfo> stripe_infos;

        // Coalesce stripe info at the source file later since that makes downstream processing much
        // easier in impl::read
        for (const size_t &stripe_idx : user_specified_stripes[src_file_idx]) {
          CUDF_EXPECTS(
            stripe_idx >= 0 && stripe_idx < per_file_metadata[src_file_idx].ff.stripes.size(),
            "Invalid stripe index");
          stripe_idxs.push_back(stripe_idx);
          stripe_infos.push_back(
            std::make_pair(&per_file_metadata[src_file_idx].ff.stripes[stripe_idx], nullptr));
          row_count += per_file_metadata[src_file_idx].ff.stripes[stripe_idx].numberOfRows;
        }
        selected_stripes_mapping.push_back(
          {static_cast<int>(src_file_idx), stripe_idxs, stripe_infos});
      }
    } else {
      row_start = std::max(row_start, 0);
      if (row_count < 0) {
        row_count = static_cast<size_type>(
          std::min<int64_t>(get_num_rows(), std::numeric_limits<size_type>::max()));
      }
      row_count = std::min(row_count, get_num_rows() - row_start);
      CUDF_EXPECTS(row_count >= 0, "Invalid row count");
      CUDF_EXPECTS(row_start <= get_num_rows(), "Invalid row start");

      size_type count = 0;
      // Iterate all source files, each source file has corelating metadata
      for (size_t src_file_idx = 0; src_file_idx < per_file_metadata.size(); ++src_file_idx) {
        std::vector<int> stripe_idxs;
        std::vector<OrcStripeInfo> stripe_infos;

        for (size_t stripe_idx = 0; stripe_idx < per_file_metadata[src_file_idx].ff.stripes.size();
             ++stripe_idx) {
          count += per_file_metadata[src_file_idx].ff.stripes[stripe_idx].numberOfRows;
          if (count > row_start || count == 0) {
            stripe_idxs.push_back(stripe_idx);
            stripe_infos.push_back(
              std::make_pair(&per_file_metadata[src_file_idx].ff.stripes[stripe_idx], nullptr));
          }
          if (count >= row_start + row_count) { break; }
        }

        selected_stripes_mapping.push_back(
          {static_cast<int>(src_file_idx), stripe_idxs, stripe_infos});
      }
    }

    // Read each stripe's stripefooter metadata
    if (not selected_stripes_mapping.empty()) {
      for (auto &mapping : selected_stripes_mapping) {
        // Resize to all stripe_info for the source level

        // Get the number of unique stripes since the same stripes could be specified more than once
        // by the user
        // size_t uniqueCount = std::unique(mapping.stripe_idx_in_source.begin(),
        // mapping.stripe_idx_in_source.end()) - mapping.stripe_idx_in_source.begin();
        per_file_metadata[mapping.source_idx].stripefooters.resize(
          mapping.stripe_idx_in_source.size());

        for (size_t i = 0; i < mapping.stripe_idx_in_source.size(); i++) {
          // int stripe_idx            = mapping.stripe_idx_in_source[i];
          const auto stripe         = mapping.stripe_info[i].first;
          const auto sf_comp_offset = stripe->offset + stripe->indexLength + stripe->dataLength;
          const auto sf_comp_length = stripe->footerLength;
          CUDF_EXPECTS(
            sf_comp_offset + sf_comp_length < per_file_metadata[mapping.source_idx].source->size(),
            "Invalid stripe information");
          const auto buffer =
            per_file_metadata[mapping.source_idx].source->host_read(sf_comp_offset, sf_comp_length);
          size_t sf_length = 0;
          auto sf_data     = per_file_metadata[mapping.source_idx].decompressor->Decompress(
            buffer->data(), sf_comp_length, &sf_length);
          ProtobufReader(sf_data, sf_length)
            .read(per_file_metadata[mapping.source_idx].stripefooters[i]);
          mapping.stripe_info[i].second = &per_file_metadata[mapping.source_idx].stripefooters[i];
        }
      }
    }

    return selected_stripes_mapping;
  }

  /**
   * @brief Adds column as per the request and saves metadata about children.
   *        Struct children are in the same level as struct, only list column
   *        children are pushed to next level.
   *
   * @param selection A vector that saves list of columns as per levels of nesting.
   * @param types A vector of schema types of columns
   * @param level current level of nesting
   * @param id current column id that needs to be added
   * @param num_lvl_child_columns number of child columns which are in the same level
   * @param has_timestamp_column True if timestamp column present and false otherwise
   *
   * @return number of columns added.
   */
  uint32_t add_column(std::vector<std::vector<orc_column_meta>> &selection,
                      std::vector<SchemaType> const &types,
                      size_t level,
                      uint32_t id,
                      uint32_t &num_lvl_child_columns,
                      bool &has_timestamp_column)
  {
    int num_cols_added = 1;
    if (level == selection.size()) { selection.push_back(std::vector<orc_column_meta>()); }
    selection[level].emplace_back(id, 0);
    int col_id = selection[level].size() - 1;
    if (types[id].kind == orc::TIMESTAMP) { has_timestamp_column = true; }
    uint32_t lvl_cols = 0;

    switch (types[id].kind) {
      case orc::LIST:
        if (not types[id].subtypes.empty()) {
          lvl_cols += 1;
          num_cols_added +=
            add_column(selection, types, level + 1, id + 1, lvl_cols, has_timestamp_column);
        }
        selection[level][col_id].num_children = lvl_cols;
        break;

      case orc::STRUCT:
        for (auto child_id : types[id].subtypes) {
          num_lvl_child_columns += 1;
          num_cols_added += add_column(
            selection, types, level, child_id, num_lvl_child_columns, has_timestamp_column);
        }
        selection[level][col_id].num_children = num_lvl_child_columns;
        break;

      default: break;
    }

    return num_cols_added;
  }

  /**
   * @brief Filters and reduces down to a selection of columns
   *
   * @param use_names List of column names to select
   * @param has_timestamp_column True if timestamp column present and false otherwise
   *
   * @return Vector of list of ORC column meta
   */
  std::vector<std::vector<orc_column_meta>> select_columns(std::vector<std::string> use_names,
                                                           bool &has_timestamp_column)
  {
    auto const &pfm = per_file_metadata[0];
    std::vector<std::vector<orc_column_meta>> selection;
    auto const num_columns = pfm.ff.types.size();
    uint32_t tmp           = 0;

    if (not use_names.empty()) {
      uint32_t index = 0;
      for (const auto &use_name : use_names) {
        bool name_found = false;
        for (uint32_t i = 0; i < num_columns; ++i, ++index) {
          if (index >= num_columns) { index = 0; }
          if (pfm.get_column_name(index) == use_name) {
            name_found = true;
            index += add_column(selection, pfm.ff.types, 0, index, tmp, has_timestamp_column);
            tmp = 0;
            break;
          }
        }
        CUDF_EXPECTS(name_found, "Unknown column name : " + std::string(use_name));
      }
    } else {
      for (uint32_t i = 1; i < num_columns;) {
        i += add_column(selection, pfm.ff.types, 0, i, tmp, has_timestamp_column);
        tmp = 0;
      }
    }

    return selection;
  }
};

rmm::device_buffer reader::impl::decompress_stripe_data(
  hostdevice_vector<gpu::ColumnDesc> &chunks,
  const std::vector<rmm::device_buffer> &stripe_data,
  const OrcDecompressor *decompressor,
  std::vector<orc_stream_info> &stream_info,
  size_t num_stripes,
  device_span<gpu::RowGroup> row_groups,
  size_t row_index_stride,
  rmm::cuda_stream_view stream)
{
  // Parse the columns' compressed info
  hostdevice_vector<gpu::CompressedStreamInfo> compinfo(0, stream_info.size(), stream);
  for (const auto &info : stream_info) {
    compinfo.insert(gpu::CompressedStreamInfo(
      static_cast<const uint8_t *>(stripe_data[info.stripe_idx].data()) + info.dst_pos,
      info.length));
  }
  compinfo.host_to_device(stream);
  gpu::ParseCompressedStripeData(compinfo.device_ptr(),
                                 compinfo.size(),
                                 decompressor->GetBlockSize(),
                                 decompressor->GetLog2MaxCompressionRatio(),
                                 stream);
  compinfo.device_to_host(stream, true);

  // Count the exact number of compressed blocks
  size_t num_compressed_blocks   = 0;
  size_t num_uncompressed_blocks = 0;
  size_t total_decomp_size       = 0;
  for (size_t i = 0; i < compinfo.size(); ++i) {
    num_compressed_blocks += compinfo[i].num_compressed_blocks;
    num_uncompressed_blocks += compinfo[i].num_uncompressed_blocks;
    total_decomp_size += compinfo[i].max_uncompressed_size;
  }
  CUDF_EXPECTS(total_decomp_size > 0, "No decompressible data found");

  rmm::device_buffer decomp_data(total_decomp_size, stream);
  rmm::device_uvector<gpu_inflate_input_s> inflate_in(
    num_compressed_blocks + num_uncompressed_blocks, stream);
  rmm::device_uvector<gpu_inflate_status_s> inflate_out(num_compressed_blocks, stream);

  // Parse again to populate the decompression input/output buffers
  size_t decomp_offset      = 0;
  uint32_t start_pos        = 0;
  uint32_t start_pos_uncomp = (uint32_t)num_compressed_blocks;
  for (size_t i = 0; i < compinfo.size(); ++i) {
    auto dst_base                 = static_cast<uint8_t *>(decomp_data.data());
    compinfo[i].uncompressed_data = dst_base + decomp_offset;
    compinfo[i].decctl            = inflate_in.data() + start_pos;
    compinfo[i].decstatus         = inflate_out.data() + start_pos;
    compinfo[i].copyctl           = inflate_in.data() + start_pos_uncomp;

    stream_info[i].dst_pos = decomp_offset;
    decomp_offset += compinfo[i].max_uncompressed_size;
    start_pos += compinfo[i].num_compressed_blocks;
    start_pos_uncomp += compinfo[i].num_uncompressed_blocks;
  }
  compinfo.host_to_device(stream);
  gpu::ParseCompressedStripeData(compinfo.device_ptr(),
                                 compinfo.size(),
                                 decompressor->GetBlockSize(),
                                 decompressor->GetLog2MaxCompressionRatio(),
                                 stream);

  // Dispatch batches of blocks to decompress
  if (num_compressed_blocks > 0) {
    switch (decompressor->GetKind()) {
      case orc::ZLIB:
        CUDA_TRY(
          gpuinflate(inflate_in.data(), inflate_out.data(), num_compressed_blocks, 0, stream));
        break;
      case orc::SNAPPY:
        CUDA_TRY(gpu_unsnap(inflate_in.data(), inflate_out.data(), num_compressed_blocks, stream));
        break;
      default: CUDF_EXPECTS(false, "Unexpected decompression dispatch"); break;
    }
  }
  if (num_uncompressed_blocks > 0) {
    CUDA_TRY(gpu_copy_uncompressed_blocks(
      inflate_in.data() + num_compressed_blocks, num_uncompressed_blocks, stream));
  }
  gpu::PostDecompressionReassemble(compinfo.device_ptr(), compinfo.size(), stream);

  // Update the stream information with the updated uncompressed info
  // TBD: We could update the value from the information we already
  // have in stream_info[], but using the gpu results also updates
  // max_uncompressed_size to the actual uncompressed size, or zero if
  // decompression failed.
  compinfo.device_to_host(stream, true);

  const size_t num_columns = chunks.size() / num_stripes;

  for (size_t i = 0; i < num_stripes; ++i) {
    for (size_t j = 0; j < num_columns; ++j) {
      auto &chunk = chunks[i * num_columns + j];
      for (int k = 0; k < gpu::CI_NUM_STREAMS; ++k) {
        if (chunk.strm_len[k] > 0 && chunk.strm_id[k] < compinfo.size()) {
          chunk.streams[k]  = compinfo[chunk.strm_id[k]].uncompressed_data;
          chunk.strm_len[k] = compinfo[chunk.strm_id[k]].max_uncompressed_size;
        }
      }
    }
  }

  if (not row_groups.empty()) {
    chunks.host_to_device(stream);
    gpu::ParseRowGroupIndex(row_groups.data(),
                            compinfo.device_ptr(),
                            chunks.device_ptr(),
                            num_columns,
                            num_stripes,
                            row_groups.size() / num_columns,
                            row_index_stride,
                            stream);
  }

  return decomp_data;
}

void reader::impl::decode_stream_data(hostdevice_vector<gpu::ColumnDesc> &chunks,
                                      size_t num_dicts,
                                      size_t skip_rows,
                                      timezone_table_view tz_table,
                                      device_span<gpu::RowGroup const> row_groups,
                                      size_t row_index_stride,
                                      std::vector<column_buffer> &out_buffers,
                                      rmm::cuda_stream_view stream)
{
  const auto num_columns = out_buffers.size();
  const auto num_stripes = chunks.size() / out_buffers.size();

  // Update chunks with pointers to column data
  for (size_t i = 0; i < num_stripes; ++i) {
    for (size_t j = 0; j < num_columns; ++j) {
      auto &chunk            = chunks[i * num_columns + j];
      chunk.column_data_base = out_buffers[j].data();
      chunk.valid_map_base   = out_buffers[j].null_mask();
    }
  }

  // Allocate global dictionary for deserializing
  rmm::device_uvector<gpu::DictionaryEntry> global_dict(num_dicts, stream);

  chunks.host_to_device(stream);
  gpu::DecodeNullsAndStringDictionaries(
    chunks.device_ptr(), global_dict.data(), num_columns, num_stripes, skip_rows, stream);
  gpu::DecodeOrcColumnData(chunks.device_ptr(),
                           global_dict.data(),
                           num_columns,
                           num_stripes,
                           skip_rows,
                           tz_table,
                           row_groups.data(),
                           row_groups.size() / num_columns,
                           row_index_stride,
                           stream);
  chunks.device_to_host(stream, true);

  for (size_t i = 0; i < num_stripes; ++i) {
    for (size_t j = 0; j < num_columns; ++j) {
      out_buffers[j].null_count() += chunks[i * num_columns + j].null_count;
    }
  }
}

/**
 * @brief Aggregate child metadata from processed parent column.
 *
 * @param chunks Vector of parent column chunks.
 * @param num_child_rows number of rows in whole child column.
 * @param child_start_row start row of each child in each stripe/chunk.
 * @param num_child_rows_per_stripe number of rows in child column per stripe/chunk.
 * @param list_col Vector of column metadata of list type parent columns.
 * @param orc_col_map Mapping between column id in orc to processing order.
 * @param number_of_stripes number of stripes being processed.
 * @param level Current nesting level being processed.
 * @param stream CUDA stream used for device memory operations and kernel launches.
 */
void reader::impl::aggregate_child_meta(hostdevice_vector<gpu::ColumnDesc> &chunks,
                                        std::vector<int32_t> &num_child_rows,
                                        std::vector<int32_t> &child_start_row,
                                        std::vector<int32_t> &num_child_rows_per_stripe,
                                        std::vector<orc_column_meta> const &list_col,
                                        std::vector<int32_t> &orc_col_map,
                                        size_t number_of_stripes,
                                        int32_t level,
                                        rmm::cuda_stream_view stream)
{
  auto num_cols               = _selected_columns[level].size();
  auto num_child_cols         = _selected_columns[level + 1].size();
  auto number_of_child_chunks = num_child_cols * number_of_stripes;

  num_child_rows.resize(_selected_columns[level + 1].size());
  std::fill(num_child_rows.begin(), num_child_rows.end(), 0);
  child_start_row.resize(number_of_child_chunks);
  num_child_rows_per_stripe.resize(number_of_child_chunks);

  chunks.device_to_host(stream, true);

  int index = 0;
  std::for_each(list_col.cbegin(), list_col.cend(), [&](auto p_col) {
    auto col_idx   = orc_col_map[p_col.id];
    auto start_row = 0;

    for (size_t i = 0; i < number_of_stripes; i++) {
      auto child_rows = chunks[i * num_cols + col_idx].num_child_rows;
      for (uint32_t j = 0; j < p_col.num_children; j++) {
        num_child_rows[index + j] += child_rows;
        num_child_rows_per_stripe[i * num_child_cols + index + j] = child_rows;
        child_start_row[i * num_child_cols + index + j]           = (i == 0) ? 0 : start_row;
      }
      start_row += child_rows;
    }
    index += p_col.num_children;
  });
}

std::unique_ptr<column> reader::impl::create_empty_column(int32_t orc_col_id,
                                                          column_name_info &schema_info,
                                                          rmm::cuda_stream_view stream)
{
  auto const schema = _metadata->get_schema(orc_col_id);
  schema_info.name  = _metadata->get_column_name(0, orc_col_id);
  auto const type   = to_type_id(schema, _use_np_dtypes, _timestamp_type.id());
  int32_t scale     = 0;
  std::vector<std::unique_ptr<column>> child_columns;
  std::unique_ptr<column> out_col = nullptr;
  switch (type) {
    case type_id::LIST:
      schema_info.children.emplace_back("offsets");
      schema_info.children.emplace_back("");
      out_col = make_lists_column(
        0,
        make_empty_column(data_type(type_id::INT32)),
        create_empty_column(
          _metadata->get_col_type(orc_col_id).subtypes[0], schema_info.children.back(), stream),
        0,
        rmm::device_buffer{0, stream},
        stream);

      break;

    case type_id::STRUCT:
      for (auto col : _metadata->get_col_type(orc_col_id).subtypes) {
        schema_info.children.emplace_back("");
        child_columns.push_back(create_empty_column(col, schema_info.children.back(), stream));
      }
      out_col =
        make_structs_column(0, std::move(child_columns), 0, rmm::device_buffer{0, stream}, stream);
      break;

    case type_id::DECIMAL64:
      scale = -static_cast<int32_t>(_metadata->get_types()[orc_col_id].scale.value_or(0));
    default: out_col = make_empty_column(data_type(type, scale));
  }

  return out_col;
}

/**
 * @brief Assemble the buffer with child columns.
 *
 * @param orc_col_id Column id in orc.
 * @param col_buffers Column buffers for columns and children.
 * @param orc_col_map Mapping between column id in orc to processing order.
 * @param level Current nesting level.
 * @param stream CUDA stream used for device memory operations and kernel launches.
 */
column_buffer &&reader::impl::assemble_buffer(int32_t orc_col_id,
                                              std::vector<std::vector<column_buffer>> &col_buffers,
                                              std::vector<std::vector<int32_t>> const &orc_col_map,
                                              int level,
                                              rmm::cuda_stream_view stream)
{
  auto const col_id = orc_col_map[level][orc_col_id];
  auto &col_buffer  = col_buffers[level][col_id];

  col_buffer.name = _metadata->get_column_name(0, orc_col_id);
  switch (col_buffer.type.id()) {
    case type_id::LIST:
      col_buffer.children.emplace_back(
        assemble_buffer(_metadata->get_col_type(orc_col_id).subtypes[0],
                        col_buffers,
                        orc_col_map,
                        level + 1,
                        stream));
      break;

    case type_id::STRUCT:
      for (auto col : _metadata->get_col_type(orc_col_id).subtypes) {
        col_buffer.children.emplace_back(
          assemble_buffer(col, col_buffers, orc_col_map, level, stream));
      }

      break;

    default: break;
  }

  return std::move(col_buffer);
}

/**
 * @brief Create columns and respective schema information from the buffer.
 *
 * @param col_buffers Column buffers for columns and children.
 * @param out_columns Vector of columns formed from column buffers.
 * @param schema_info Vector of schema information formed from column buffers.
 * @param orc_col_map Mapping between column id in orc to processing order.
 * @param stream CUDA stream used for device memory operations and kernel launches.
 */
void reader::impl::create_columns(std::vector<std::vector<column_buffer>> &col_buffers,
                                  std::vector<std::unique_ptr<column>> &out_columns,
                                  std::vector<column_name_info> &schema_info,
                                  std::vector<std::vector<int>> const &orc_col_map,
                                  rmm::cuda_stream_view stream)
{
  for (size_t i = 0; i < _selected_columns[0].size();) {
    auto const &col_meta = _selected_columns[0][i];
    schema_info.emplace_back("");
    auto col_buffer = assemble_buffer(col_meta.id, col_buffers, orc_col_map, 0, stream);
    out_columns.emplace_back(make_column(col_buffer, &schema_info.back(), stream, _mr));
    i += (col_buffers[0][i].type.id() == type_id::STRUCT) ? col_meta.num_children + 1 : 1;
  }
}

reader::impl::impl(std::vector<std::unique_ptr<datasource>> &&sources,
                   orc_reader_options const &options,
                   rmm::mr::device_memory_resource *mr)
  : _mr(mr), _sources(std::move(sources))
{
  // Open and parse the source(s) dataset metadata
  _metadata = std::make_unique<aggregate_orc_metadata>(_sources);

  // Select only columns required by the options
  _selected_columns = _metadata->select_columns(options.get_columns(), _has_timestamp_column);

  // Override output timestamp resolution if requested
  if (options.get_timestamp_type().id() != type_id::EMPTY) {
    _timestamp_type = options.get_timestamp_type();
  }

  // Enable or disable attempt to use row index for parsing
  _use_index = options.is_enabled_use_index();

  // Enable or disable the conversion to numpy-compatible dtypes
  _use_np_dtypes = options.is_enabled_use_np_dtypes();
}

table_with_metadata reader::impl::read(size_type skip_rows,
                                       size_type num_rows,
                                       const std::vector<std::vector<size_type>> &stripes,
                                       rmm::cuda_stream_view stream)
{
  std::vector<std::unique_ptr<column>> out_columns;
  std::vector<std::vector<int>> orc_col_id_map(_selected_columns.size());
  std::vector<std::vector<column_buffer>> out_buffers(_selected_columns.size());
  std::vector<std::vector<int32_t>> orc_col_map;
  std::vector<column_name_info> schema_info;
  std::vector<std::vector<rmm::device_buffer>> lvl_stripe_data(_selected_columns.size());
  table_metadata out_metadata;

  // TBD : Need to update num_rows for later set of levels

  // There are no columns in table
  if (_selected_columns.size() == 0) return {std::make_unique<table>(), std::move(out_metadata)};

  // Select only stripes required (aka row groups)
  const auto selected_stripes = _metadata->select_stripes(stripes, skip_rows, num_rows);

  std::vector<int32_t> num_child_rows;   // number of child rows of a list column
  std::vector<int32_t> child_start_row;  // start row of a child column in a stripe
  std::vector<int32_t> num_child_rows_per_stripe;
  // Iterates through levels of nested columns, struct columns and its children will be
  // in the same level, list column children will be in 1 level down compared to parent.
  for (size_t level = 0; level < _selected_columns.size(); level++) {
    auto &selected_columns = _selected_columns[level];
    // Association between each ORC column and its cudf::column
    orc_col_map.emplace_back(_metadata->get_num_cols(), -1);
    std::vector<orc_column_meta> list_col;

    // Get a list of column data types
    std::vector<data_type> column_types;
    for (auto &col : selected_columns) {
      auto col_type =
        to_type_id(_metadata->get_col_type(col.id), _use_np_dtypes, _timestamp_type.id());
      CUDF_EXPECTS(col_type != type_id::EMPTY, "Unknown type");
      // Remove this once we support Decimal128 data type
      CUDF_EXPECTS(
        (col_type != type_id::DECIMAL64) or (_metadata->get_types()[col.id].precision <= 18),
        "Decimal data has precision > 18, Decimal64 data type doesn't support it.");
      if (col_type == type_id::DECIMAL64) {
        // sign of the scale is changed since cuDF follows c++ libraries like CNL
        // which uses negative scaling, but liborc and other libraries
        // follow positive scaling.
        auto const scale = -static_cast<int32_t>(_metadata->get_types()[col.id].scale.value_or(0));
        column_types.emplace_back(col_type, scale);
      } else {
        column_types.emplace_back(col_type);
      }

      // Map each ORC column to its column
      orc_col_map[level][col.id] = column_types.size() - 1;
      if (col_type == type_id::LIST) list_col.emplace_back(col);
    }

    // If no rows or stripes to read, return empty columns
    if (num_rows <= 0 || selected_stripes.empty()) {
      for (size_t i = 0; i < _selected_columns[0].size();) {
        auto const &col_meta = _selected_columns[0][i];
        auto const schema    = _metadata->get_schema(col_meta.id);
        schema_info.emplace_back("");
        out_columns.push_back(
          std::move(create_empty_column(col_meta.id, schema_info.back(), stream)));
        // Since struct children will be in the same level, have to skip them.
        i += (schema.kind == orc::STRUCT) ? col_meta.num_children + 1 : 1;
      }
      break;
    } else {
      // Get the total number of stripes across all input files.
      size_t total_num_stripes = 0;
      for (const auto &stripe_source_mapping : selected_stripes) {
        total_num_stripes += stripe_source_mapping.stripe_idx_in_source.size();
      }
      const auto num_columns = selected_columns.size();
      const auto num_chunks  = total_num_stripes * num_columns;
      hostdevice_vector<gpu::ColumnDesc> chunks(num_chunks, stream);
      memset(chunks.host_ptr(), 0, chunks.memory_size());

      const bool use_index =
        (_use_index == true) &&
        // Only use if we don't have much work with complete columns & stripes
        // TODO: Consider nrows, gpu, and tune the threshold
        (num_rows > _metadata->get_row_index_stride() && !(_metadata->get_row_index_stride() & 7) &&
         _metadata->get_row_index_stride() > 0 && num_columns * total_num_stripes < 8 * 128) &&
        // Only use if first row is aligned to a stripe boundary
        // TODO: Fix logic to handle unaligned rows
        (skip_rows == 0);

      // Logically view streams as columns
      std::vector<orc_stream_info> stream_info;

      // Tracker for eventually deallocating compressed and uncompressed data
      auto &stripe_data = lvl_stripe_data[level];

      size_t stripe_start_row = 0;
      size_t num_dict_entries = 0;
      size_t num_rowgroups    = 0;
      int stripe_idx          = 0;

      for (auto &stripe_source_mapping : selected_stripes) {
        // Iterate through the source files selected stripes
        for (auto &stripe : stripe_source_mapping.stripe_info) {
          const auto stripe_info   = stripe.first;
          const auto stripe_footer = stripe.second;

          auto stream_count          = stream_info.size();
          const auto total_data_size = gather_stream_info(stripe_idx,
                                                          stripe_info,
                                                          stripe_footer,
                                                          orc_col_map[level],
                                                          selected_columns,
                                                          _metadata->get_types(),
                                                          use_index,
                                                          &num_dict_entries,
                                                          chunks,
                                                          stream_info);

          CUDF_EXPECTS(total_data_size > 0, "Expected streams data within stripe");

          stripe_data.emplace_back(total_data_size, stream);
          auto dst_base = static_cast<uint8_t *>(stripe_data.back().data());

          // Coalesce consecutive streams into one read
          while (stream_count < stream_info.size()) {
            const auto d_dst  = dst_base + stream_info[stream_count].dst_pos;
            const auto offset = stream_info[stream_count].offset;
            auto len          = stream_info[stream_count].length;
            stream_count++;

            while (stream_count < stream_info.size() &&
                   stream_info[stream_count].offset == offset + len) {
              len += stream_info[stream_count].length;
              stream_count++;
            }
            if (_metadata->per_file_metadata[stripe_source_mapping.source_idx]
                  .source->is_device_read_preferred(len)) {
              CUDF_EXPECTS(
                _metadata->per_file_metadata[stripe_source_mapping.source_idx].source->device_read(
                  offset, len, d_dst, stream) == len,
                "Unexpected discrepancy in bytes read.");
            } else {
              const auto buffer =
                _metadata->per_file_metadata[stripe_source_mapping.source_idx].source->host_read(
                  offset, len);
              CUDF_EXPECTS(buffer->size() == len, "Unexpected discrepancy in bytes read.");
              CUDA_TRY(hipMemcpyAsync(
                d_dst, buffer->data(), len, hipMemcpyHostToDevice, stream.value()));
              stream.synchronize();
            }
          }

          // Update chunks to reference streams pointers
          uint32_t max_num_rows = 0;
          for (size_t col_idx = 0; col_idx < num_columns; col_idx++) {
            auto &chunk = chunks[stripe_idx * num_columns + col_idx];
            chunk.start_row =
              (level == 0) ? stripe_start_row : child_start_row[stripe_idx * num_columns + col_idx];
            chunk.num_rows = (level == 0)
                               ? stripe_info->numberOfRows
                               : num_child_rows_per_stripe[stripe_idx * num_columns + col_idx];
            chunk.column_num_rows = (level == 0) ? num_rows : num_child_rows[col_idx];
            chunk.encoding_kind   = stripe_footer->columns[selected_columns[col_idx].id].kind;
            chunk.type_kind       = _metadata->per_file_metadata[stripe_source_mapping.source_idx]
                                .ff.types[selected_columns[col_idx].id]
                                .kind;
            chunk.decimal_scale = _metadata->per_file_metadata[stripe_source_mapping.source_idx]
                                    .ff.types[selected_columns[col_idx].id]
                                    .scale.value_or(0);
            chunk.rowgroup_id = num_rowgroups;
            chunk.dtype_len   = (column_types[col_idx].id() == type_id::STRING)
                                ? sizeof(std::pair<const char *, size_t>)
                                : ((column_types[col_idx].id() == type_id::LIST) or
                                   (column_types[col_idx].id() == type_id::STRUCT))
                                    ? sizeof(int32_t)
                                    : cudf::size_of(column_types[col_idx]);
            if (chunk.type_kind == orc::TIMESTAMP) {
              chunk.ts_clock_rate = to_clockrate(_timestamp_type.id());
            }
            for (int k = 0; k < gpu::CI_NUM_STREAMS; k++) {
              chunk.streams[k] = dst_base + stream_info[chunk.strm_id[k]].dst_pos;
            }
            if (level > 0 and max_num_rows > chunk.num_rows) { max_num_rows = chunk.num_rows; }
          }
          auto num_rows_per_stripe = (level == 0) ? stripe_info->numberOfRows : max_num_rows;
          stripe_start_row += num_rows_per_stripe;
          if (use_index) {
            num_rowgroups += (num_rows_per_stripe + _metadata->get_row_index_stride() - 1) /
                             _metadata->get_row_index_stride();
          }

          stripe_idx++;
        }
      }

      // Process dataset chunk pages into output columns
      if (stripe_data.size() != 0) {
        // Setup row group descriptors if using indexes
        auto row_groups = cudf::detail::make_zeroed_device_uvector_sync<gpu::RowGroup>(
          num_rowgroups * num_columns, stream, _mr);
        if (_metadata->per_file_metadata[0].ps.compression != orc::NONE) {
          auto decomp_data =
            decompress_stripe_data(chunks,
                                   stripe_data,
                                   _metadata->per_file_metadata[0].decompressor.get(),
                                   stream_info,
                                   total_num_stripes,
                                   row_groups,
                                   _metadata->get_row_index_stride(),
                                   stream);
          stripe_data.clear();
          stripe_data.push_back(std::move(decomp_data));
        } else {
          if (not row_groups.is_empty()) {
            chunks.host_to_device(stream);
            gpu::ParseRowGroupIndex(row_groups.data(),
                                    nullptr,
                                    chunks.device_ptr(),
                                    num_columns,
                                    total_num_stripes,
                                    num_rowgroups,
                                    _metadata->get_row_index_stride(),
                                    stream);
          }
        }

        // Setup table for converting timestamp columns from local to UTC time
        auto const tz_table =
          _has_timestamp_column
            ? build_timezone_transition_table(
                selected_stripes[0].stripe_info[0].second->writerTimezone, stream)
            : timezone_table{};

        for (size_t i = 0; i < column_types.size(); ++i) {
          bool is_nullable = false;
          for (size_t j = 0; j < total_num_stripes; ++j) {
            if (chunks[j * num_columns + i].strm_len[gpu::CI_PRESENT] != 0) {
              is_nullable = true;
              break;
            }
          }
          auto is_list_type = (column_types[i].id() == type_id::LIST);
          auto n_rows       = (level == 0) ? num_rows : num_child_rows[i];
          n_rows += is_list_type;
          out_buffers[level].emplace_back(column_types[i], n_rows, is_nullable, stream, _mr);
        }

        decode_stream_data(chunks,
                           num_dict_entries,
                           skip_rows,
                           tz_table.view(),
                           row_groups,
                           _metadata->get_row_index_stride(),
                           out_buffers[level],
                           stream);

        // Extract information to process child columns
        if (list_col.size()) {
          aggregate_child_meta(chunks,
                               num_child_rows,
                               child_start_row,
                               num_child_rows_per_stripe,
                               list_col,
                               orc_col_map[level],
                               selected_stripes.size(),
                               level,
                               stream);
        }
      }

      // TO-DO: Replace this with exclusive scan that will work on all list columns for that level
      if (list_col.size()) {
        std::for_each(
          out_buffers[level].begin(), out_buffers[level].end(), [stream](auto &out_buffer) {
            if (out_buffer.type.id() == type_id::LIST) {
              auto data = static_cast<size_type *>(out_buffer.data());
              thrust::exclusive_scan(rmm::exec_policy(stream), data, data + out_buffer.size, data);
            }
          });
      }
    }
  }

  // If out_columns is empty, then create columns from buffer.
  if (!out_columns.size()) {
    create_columns(out_buffers, out_columns, schema_info, orc_col_map, stream);
  }

  // Return column names (must match order of returned columns)
  out_metadata.column_names.resize(schema_info.size());
  for (size_t i = 0; i < schema_info.size(); i++) {
    out_metadata.column_names[i] = schema_info[i].name;
  }
  out_metadata.schema_info = std::move(schema_info);

  // XXX: Review question. Should metadata from all input files be included here as I am doing
  // or just a single input file? Return user metadata
  for (const auto &meta : _metadata->per_file_metadata) {
    for (const auto &kv : meta.ff.metadata) { out_metadata.user_data.insert({kv.name, kv.value}); }
  }

  return {std::make_unique<table>(std::move(out_columns)), std::move(out_metadata)};
}

// Forward to implementation
reader::reader(std::vector<std::string> const &filepaths,
               orc_reader_options const &options,
               rmm::cuda_stream_view stream,
               rmm::mr::device_memory_resource *mr)
{
  _impl = std::make_unique<impl>(datasource::create(filepaths), options, mr);
}

// Forward to implementation
reader::reader(std::vector<std::unique_ptr<cudf::io::datasource>> &&sources,
               orc_reader_options const &options,
               rmm::cuda_stream_view stream,
               rmm::mr::device_memory_resource *mr)
{
  _impl = std::make_unique<impl>(std::move(sources), options, mr);
}

// Destructor within this translation unit
reader::~reader() = default;

// Forward to implementation
table_with_metadata reader::read(orc_reader_options const &options, rmm::cuda_stream_view stream)
{
  return _impl->read(
    options.get_skip_rows(), options.get_num_rows(), options.get_stripes(), stream);
}
}  // namespace orc
}  // namespace detail
}  // namespace io
}  // namespace cudf
