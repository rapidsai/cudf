#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019-2021, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

/**
 * @file writer_impl.cu
 * @brief cuDF-IO ORC writer class implementation
 */

#include "writer_impl.hpp"

#include <cudf/null_mask.hpp>
#include <cudf/strings/strings_column_view.hpp>

#include <rmm/cuda_stream_view.hpp>
#include <rmm/device_buffer.hpp>
#include <rmm/device_vector.hpp>

#include <algorithm>
#include <cstring>
#include <utility>

namespace cudf {
namespace io {
namespace detail {
namespace orc {
using namespace cudf::io::orc;
using namespace cudf::io;

struct row_group_index_info {
  int32_t pos       = -1;  // Position
  int32_t blk_pos   = -1;  // Block Position
  int32_t comp_pos  = -1;  // Compressed Position
  int32_t comp_size = -1;  // Compressed size
};

namespace {
/**
 * @brief Helper for pinned host memory
 */
template <typename T>
using pinned_buffer = std::unique_ptr<T, decltype(&hipHostFree)>;

/**
 * @brief Function that translates GDF compression to ORC compression
 */
orc::CompressionKind to_orc_compression(compression_type compression)
{
  switch (compression) {
    case compression_type::AUTO:
    case compression_type::SNAPPY: return orc::CompressionKind::SNAPPY;
    case compression_type::NONE: return orc::CompressionKind::NONE;
    default: CUDF_EXPECTS(false, "Unsupported compression type"); return orc::CompressionKind::NONE;
  }
}

/**
 * @brief Function that translates GDF dtype to ORC datatype
 */
constexpr orc::TypeKind to_orc_type(cudf::type_id id)
{
  switch (id) {
    case cudf::type_id::INT8: return TypeKind::BYTE;
    case cudf::type_id::INT16: return TypeKind::SHORT;
    case cudf::type_id::INT32: return TypeKind::INT;
    case cudf::type_id::INT64: return TypeKind::LONG;
    case cudf::type_id::FLOAT32: return TypeKind::FLOAT;
    case cudf::type_id::FLOAT64: return TypeKind::DOUBLE;
    case cudf::type_id::BOOL8: return TypeKind::BOOLEAN;
    case cudf::type_id::TIMESTAMP_DAYS: return TypeKind::DATE;
    case cudf::type_id::TIMESTAMP_SECONDS:
    case cudf::type_id::TIMESTAMP_MICROSECONDS:
    case cudf::type_id::TIMESTAMP_MILLISECONDS:
    case cudf::type_id::TIMESTAMP_NANOSECONDS: return TypeKind::TIMESTAMP;
    case cudf::type_id::STRING: return TypeKind::STRING;
    default: return TypeKind::INVALID_TYPE_KIND;
  }
}

/**
 * @brief Function that translates time unit to nanoscale multiple
 */
template <typename T>
constexpr T to_clockscale(cudf::type_id timestamp_id)
{
  switch (timestamp_id) {
    case cudf::type_id::TIMESTAMP_SECONDS: return 9;
    case cudf::type_id::TIMESTAMP_MILLISECONDS: return 6;
    case cudf::type_id::TIMESTAMP_MICROSECONDS: return 3;
    case cudf::type_id::TIMESTAMP_NANOSECONDS:
    default: return 0;
  }
}

}  // namespace

/**
 * @brief Helper kernel for converting string data/offsets into nvstrdesc
 * REMOVEME: Once we eliminate the legacy readers/writers, the kernels could be
 * made to use the native offset+data layout.
 */
__global__ void stringdata_to_nvstrdesc(gpu::nvstrdesc_s *dst,
                                        const size_type *offsets,
                                        const char *strdata,
                                        const uint32_t *nulls,
                                        const size_type column_offset,
                                        size_type column_size)
{
  size_type row = blockIdx.x * blockDim.x + threadIdx.x;
  if (row < column_size) {
    uint32_t is_valid = (nulls != nullptr)
                          ? (nulls[(row + column_offset) / 32] >> ((row + column_offset) % 32)) & 1
                          : 1;
    size_t count;
    const char *ptr;
    if (is_valid) {
      size_type cur  = offsets[row];
      size_type next = offsets[row + 1];
      ptr            = strdata + cur;
      count          = (next > cur) ? next - cur : 0;
    } else {
      ptr   = nullptr;
      count = 0;
    }
    dst[row].ptr   = ptr;
    dst[row].count = count;
  }
}

/**
 * @brief Helper class that adds ORC-specific column info
 */
class orc_column_view {
 public:
  /**
   * @brief Constructor that extracts out the string position + length pairs
   * for building dictionaries for string columns
   */
  explicit orc_column_view(size_t id,
                           size_t str_id,
                           column_view const &col,
                           const table_metadata *metadata,
                           rmm::cuda_stream_view stream)
    : _id(id),
      _str_id(str_id),
      _string_type(col.type().id() == type_id::STRING),
      _type_width(_string_type ? 0 : cudf::size_of(col.type())),
      _data_count(col.size()),
      _null_count(col.null_count()),
      _data(col.head<uint8_t>() + col.offset() * _type_width),
      _nulls(col.nullable() ? col.null_mask() : nullptr),
      _column_offset(col.offset()),
      _clockscale(to_clockscale<uint8_t>(col.type().id())),
      _type_kind(to_orc_type(col.type().id()))
  {
    if (_string_type && _data_count > 0) {
      strings_column_view view{col};
      _indexes = rmm::device_buffer(_data_count * sizeof(gpu::nvstrdesc_s), stream);

      stringdata_to_nvstrdesc<<<((_data_count - 1) >> 8) + 1, 256, 0, stream.value()>>>(
        static_cast<gpu::nvstrdesc_s *>(_indexes.data()),
        view.offsets().data<size_type>() + view.offset(),
        view.chars().data<char>(),
        _nulls,
        _column_offset,
        _data_count);
      _data = _indexes.data();

      stream.synchronize();
    }
    // Generating default name if name isn't present in metadata
    if (metadata && _id < metadata->column_names.size()) {
      _name = metadata->column_names[_id];
    } else {
      _name = "_col" + std::to_string(_id);
    }
  }

  auto is_string() const noexcept { return _string_type; }
  void set_dict_stride(size_t stride) noexcept { dict_stride = stride; }
  auto get_dict_stride() const noexcept { return dict_stride; }

  /**
   * @brief Function that associates an existing dictionary chunk allocation
   */
  void attach_dict_chunk(gpu::DictionaryChunk *host_dict, gpu::DictionaryChunk *dev_dict)
  {
    dict   = host_dict;
    d_dict = dev_dict;
  }
  auto host_dict_chunk(size_t rowgroup)
  {
    assert(_string_type);
    return &dict[rowgroup * dict_stride + _str_id];
  }
  auto device_dict_chunk() const { return d_dict; }

  /**
   * @brief Function that associates an existing stripe dictionary allocation
   */
  void attach_stripe_dict(gpu::StripeDictionary *host_stripe_dict,
                          gpu::StripeDictionary *dev_stripe_dict)
  {
    stripe_dict   = host_stripe_dict;
    d_stripe_dict = dev_stripe_dict;
  }
  auto host_stripe_dict(size_t stripe) const
  {
    assert(_string_type);
    return &stripe_dict[stripe * dict_stride + _str_id];
  }
  auto device_stripe_dict() const { return d_stripe_dict; }

  size_t type_width() const noexcept { return _type_width; }
  size_t data_count() const noexcept { return _data_count; }
  size_t null_count() const noexcept { return _null_count; }
  bool nullable() const noexcept { return (_nulls != nullptr); }
  void const *data() const noexcept { return _data; }
  uint32_t const *nulls() const noexcept { return _nulls; }
  size_type column_offset() const noexcept { return _column_offset; }
  uint8_t clockscale() const noexcept { return _clockscale; }

  void set_orc_encoding(ColumnEncodingKind e) { _encoding_kind = e; }
  auto orc_kind() const noexcept { return _type_kind; }
  auto orc_encoding() const noexcept { return _encoding_kind; }
  auto orc_name() const noexcept { return _name; }

 private:
  // Identifier within set of columns and string columns, respectively
  size_t _id        = 0;
  size_t _str_id    = 0;
  bool _string_type = false;

  size_t _type_width       = 0;
  size_t _data_count       = 0;
  size_t _null_count       = 0;
  void const *_data        = nullptr;
  uint32_t const *_nulls   = nullptr;
  size_type _column_offset = 0;
  uint8_t _clockscale      = 0;

  // ORC-related members
  std::string _name{};
  TypeKind _type_kind;
  ColumnEncodingKind _encoding_kind;

  // String dictionary-related members
  rmm::device_buffer _indexes;
  size_t dict_stride                       = 0;
  gpu::DictionaryChunk const *dict         = nullptr;
  gpu::StripeDictionary const *stripe_dict = nullptr;
  gpu::DictionaryChunk *d_dict             = nullptr;
  gpu::StripeDictionary *d_stripe_dict     = nullptr;
};

void writer::impl::init_dictionaries(orc_column_view *columns,
                                     size_t num_rows,
                                     std::vector<int> const &str_col_ids,
                                     uint32_t *dict_data,
                                     uint32_t *dict_index,
                                     hostdevice_vector<gpu::DictionaryChunk> &dict)
{
  const size_t num_rowgroups = dict.size() / str_col_ids.size();

  // Setup per-rowgroup dictionary indexes for each dictionary-aware column
  for (size_t i = 0; i < str_col_ids.size(); ++i) {
    auto &str_column = columns[str_col_ids[i]];
    str_column.set_dict_stride(str_col_ids.size());
    str_column.attach_dict_chunk(dict.host_ptr(), dict.device_ptr());

    for (size_t g = 0; g < num_rowgroups; g++) {
      auto *ck              = &dict[g * str_col_ids.size() + i];
      ck->valid_map_base    = str_column.nulls();
      ck->column_offset     = str_column.column_offset();
      ck->column_data_base  = str_column.data();
      ck->dict_data         = dict_data + i * num_rows + g * row_index_stride_;
      ck->dict_index        = dict_index + i * num_rows;  // Indexed by abs row
      ck->start_row         = g * row_index_stride_;
      ck->num_rows          = std::min<uint32_t>(row_index_stride_,
                                        std::max<int>(str_column.data_count() - ck->start_row, 0));
      ck->num_strings       = 0;
      ck->string_char_count = 0;
      ck->num_dict_strings  = 0;
      ck->dict_char_count   = 0;
    }
  }

  dict.host_to_device(stream);
  gpu::InitDictionaryIndices(dict.device_ptr(), str_col_ids.size(), num_rowgroups, stream);
  dict.device_to_host(stream, true);
}

void writer::impl::build_dictionaries(orc_column_view *columns,
                                      size_t num_rows,
                                      std::vector<int> const &str_col_ids,
                                      std::vector<uint32_t> const &stripe_list,
                                      hostdevice_vector<gpu::DictionaryChunk> const &dict,
                                      uint32_t *dict_index,
                                      hostdevice_vector<gpu::StripeDictionary> &stripe_dict)
{
  const auto num_rowgroups = dict.size() / str_col_ids.size();

  for (size_t i = 0; i < str_col_ids.size(); i++) {
    size_t direct_cost = 0, dict_cost = 0;
    auto &str_column = columns[str_col_ids[i]];
    str_column.attach_stripe_dict(stripe_dict.host_ptr(), stripe_dict.device_ptr());

    for (size_t j = 0, g = 0; j < stripe_list.size(); j++) {
      const auto num_chunks = stripe_list[j];
      auto *sd              = &stripe_dict[j * str_col_ids.size() + i];
      sd->column_data_base  = str_column.host_dict_chunk(0)->column_data_base;
      sd->dict_data         = str_column.host_dict_chunk(g)->dict_data;
      sd->dict_index        = dict_index + i * num_rows;  // Indexed by abs row
      sd->column_id         = str_col_ids[i];
      sd->start_chunk       = (uint32_t)g;
      sd->num_chunks        = num_chunks;
      sd->num_strings       = 0;
      sd->dict_char_count   = 0;
      for (size_t k = g; k < g + num_chunks; k++) {
        const auto &dt = dict[k * str_col_ids.size() + i];
        sd->num_strings += dt.num_dict_strings;
        direct_cost += dt.string_char_count;
        dict_cost += dt.dict_char_count + dt.num_dict_strings;
      }

      g += num_chunks;
    }

    // Early disable of dictionary if it doesn't look good at the chunk level
    if (enable_dictionary_ && dict_cost >= direct_cost) {
      for (size_t j = 0; j < stripe_list.size(); j++) {
        stripe_dict[j * str_col_ids.size() + i].dict_data = nullptr;
      }
    }
  }

  stripe_dict.host_to_device(stream);
  gpu::BuildStripeDictionaries(stripe_dict.device_ptr(),
                               stripe_dict.host_ptr(),
                               dict.device_ptr(),
                               stripe_list.size(),
                               num_rowgroups,
                               str_col_ids.size(),
                               stream);
  stripe_dict.device_to_host(stream, true);
}

std::vector<Stream> writer::impl::gather_streams(orc_column_view *columns,
                                                 size_t num_columns,
                                                 size_t num_rows,
                                                 std::vector<uint32_t> const &stripe_list,
                                                 std::vector<int32_t> &strm_ids)
{
  // First n + 1 streams are row index streams, including 'column 0'
  std::vector<Stream> streams;
  streams.resize(num_columns + 1);
  streams[0].column = 0;
  streams[0].kind   = ROW_INDEX;
  streams[0].length = 0;

  for (size_t i = 0; i < num_columns; ++i) {
    TypeKind kind                    = columns[i].orc_kind();
    StreamKind data_kind             = DATA;
    StreamKind data2_kind            = LENGTH;
    ColumnEncodingKind encoding_kind = DIRECT;

    int64_t present_stream_size = 0;
    int64_t data_stream_size    = 0;
    int64_t data2_stream_size   = 0;
    int64_t dict_stream_size    = 0;
    bool is_nullable;

    if (single_write_mode) {
      is_nullable = (columns[i].nullable() || columns[i].data_count() < num_rows);
    } else {
      is_nullable = (i < user_metadata_with_nullability.column_nullable.size())
                      ? user_metadata_with_nullability.column_nullable[i]
                      : true;
    }
    if (is_nullable) {
      present_stream_size = ((row_index_stride_ + 7) >> 3);
      present_stream_size += (present_stream_size + 0x7f) >> 7;
    }

    switch (kind) {
      case TypeKind::BOOLEAN:
        data_stream_size = div_rowgroups_by<int64_t>(1024) * (128 + 1);
        encoding_kind    = DIRECT;
        break;
      case TypeKind::BYTE:
        data_stream_size = div_rowgroups_by<int64_t>(128) * (128 + 1);
        encoding_kind    = DIRECT;
        break;
      case TypeKind::SHORT:
        data_stream_size = div_rowgroups_by<int64_t>(512) * (512 * 2 + 2);
        encoding_kind    = DIRECT_V2;
        break;
      case TypeKind::FLOAT:
        // Pass through if no nulls (no RLE encoding for floating point)
        data_stream_size = (columns[i].null_count() != 0)
                             ? div_rowgroups_by<int64_t>(512) * (512 * 4 + 2)
                             : INT64_C(-1);
        encoding_kind = DIRECT;
        break;
      case TypeKind::INT:
      case TypeKind::DATE:
        data_stream_size = div_rowgroups_by<int64_t>(512) * (512 * 4 + 2);
        encoding_kind    = DIRECT_V2;
        break;
      case TypeKind::DOUBLE:
        // Pass through if no nulls (no RLE encoding for floating point)
        data_stream_size = (columns[i].null_count() != 0)
                             ? div_rowgroups_by<int64_t>(512) * (512 * 8 + 2)
                             : INT64_C(-1);
        encoding_kind = DIRECT;
        break;
      case TypeKind::LONG:
        data_stream_size = div_rowgroups_by<int64_t>(512) * (512 * 8 + 2);
        encoding_kind    = DIRECT_V2;
        break;
      case TypeKind::STRING: {
        bool enable_dict           = enable_dictionary_;
        size_t direct_data_size    = 0;
        size_t dict_data_size      = 0;
        size_t dict_strings        = 0;
        size_t dict_lengths_div512 = 0;
        for (size_t stripe = 0, g = 0; stripe < stripe_list.size(); stripe++) {
          const auto sd = columns[i].host_stripe_dict(stripe);
          enable_dict   = (enable_dict && sd->dict_data != nullptr);
          if (enable_dict) {
            dict_strings += sd->num_strings;
            dict_lengths_div512 += (sd->num_strings + 0x1ff) >> 9;
            dict_data_size += sd->dict_char_count;
          }

          for (uint32_t k = 0; k < stripe_list[stripe]; k++, g++) {
            direct_data_size += columns[i].host_dict_chunk(g)->string_char_count;
          }
        }
        if (enable_dict) {
          uint32_t dict_bits = 0;
          for (dict_bits = 1; dict_bits < 32; dict_bits <<= 1) {
            if (dict_strings <= (1ull << dict_bits)) break;
          }
          const auto valid_count = columns[i].data_count() - columns[i].null_count();
          dict_data_size += (dict_bits * valid_count + 7) >> 3;
        }

        // Decide between direct or dictionary encoding
        if (enable_dict && dict_data_size < direct_data_size) {
          data_stream_size  = div_rowgroups_by<int64_t>(512) * (512 * 4 + 2);
          data2_stream_size = dict_lengths_div512 * (512 * 4 + 2);
          dict_stream_size  = std::max<size_t>(dict_data_size, 1);
          encoding_kind     = DICTIONARY_V2;
        } else {
          data_stream_size  = std::max<size_t>(direct_data_size, 1);
          data2_stream_size = div_rowgroups_by<int64_t>(512) * (512 * 4 + 2);
          encoding_kind     = DIRECT_V2;
        }
        break;
      }
      case TypeKind::TIMESTAMP:
        data_stream_size  = ((row_index_stride_ + 0x1ff) >> 9) * (512 * 4 + 2);
        data2_stream_size = data_stream_size;
        data2_kind        = SECONDARY;
        encoding_kind     = DIRECT_V2;
        break;
      default: CUDF_FAIL("Unsupported ORC type kind");
    }

    // Initialize the column's metadata
    columns[i].set_orc_encoding(encoding_kind);

    // Initialize the column's index stream
    const auto id      = static_cast<uint32_t>(1 + i);
    streams[id].column = id;
    streams[id].kind   = ROW_INDEX;
    streams[id].length = 0;

    // Initialize the column's data stream(s)
    const auto base = i * gpu::CI_NUM_STREAMS;
    if (present_stream_size != 0) {
      auto len                         = static_cast<uint64_t>(present_stream_size);
      strm_ids[base + gpu::CI_PRESENT] = streams.size();
      streams.push_back(orc::Stream{PRESENT, id, len});
    }
    if (data_stream_size != 0) {
      auto len                      = static_cast<uint64_t>(std::max<int64_t>(data_stream_size, 0));
      strm_ids[base + gpu::CI_DATA] = streams.size();
      streams.push_back(orc::Stream{data_kind, id, len});
    }
    if (data2_stream_size != 0) {
      auto len = static_cast<uint64_t>(std::max<int64_t>(data2_stream_size, 0));
      strm_ids[base + gpu::CI_DATA2] = streams.size();
      streams.push_back(orc::Stream{data2_kind, id, len});
    }
    if (dict_stream_size != 0) {
      auto len                            = static_cast<uint64_t>(dict_stream_size);
      strm_ids[base + gpu::CI_DICTIONARY] = streams.size();
      streams.push_back(orc::Stream{DICTIONARY_DATA, id, len});
    }
  }

  return streams;
}

struct segmented_valid_cnt_input {
  bitmask_type const *mask;
  std::vector<size_type> indices;
};

rmm::device_buffer writer::impl::encode_columns(orc_column_view *columns,
                                                size_t num_columns,
                                                size_t num_rows,
                                                size_t num_rowgroups,
                                                std::vector<int> const &str_col_ids,
                                                std::vector<uint32_t> const &stripe_list,
                                                std::vector<Stream> const &streams,
                                                std::vector<int32_t> const &strm_ids,
                                                hostdevice_vector<gpu::EncChunk> &chunks)
{
  // Allocate combined buffer for RLE data and string data output
  std::vector<size_t> strm_offsets(streams.size());
  size_t str_data_size = 0;
  auto output          = [&]() {
    size_t rle_data_size = 0;
    for (size_t i = 0; i < streams.size(); ++i) {
      const auto &stream = streams[i];
      const auto &column = columns[stream.column - 1];

      if (((stream.kind == DICTIONARY_DATA || stream.kind == LENGTH) &&
           (column.orc_encoding() == DICTIONARY_V2)) ||
          ((stream.kind == DATA) &&
           (column.orc_kind() == TypeKind::STRING && column.orc_encoding() == DIRECT_V2))) {
        strm_offsets[i] = str_data_size;
        str_data_size += stream.length;
      } else {
        strm_offsets[i] = rle_data_size;
        rle_data_size += (stream.length * num_rowgroups + 7) & ~7;
      }
    }
    str_data_size = (str_data_size + 7) & ~7;

    return rmm::device_buffer(rle_data_size + str_data_size, stream);
  }();
  auto dst_base = static_cast<uint8_t *>(output.data());

  // Initialize column chunks' descriptions
  size_t stripe_start = 0;
  size_t stripe_id    = 0;
  std::map<size_type, segmented_valid_cnt_input> validity_check_inputs;
  for (size_t j = 0; j < num_rowgroups; j++) {
    for (size_t i = 0; i < num_columns; i++) {
      auto *ck          = &chunks[j * num_columns + i];
      ck->start_row     = (j * row_index_stride_);
      ck->num_rows      = std::min<uint32_t>(row_index_stride_, num_rows - ck->start_row);
      ck->valid_rows    = columns[i].data_count();
      ck->encoding_kind = columns[i].orc_encoding();
      ck->type_kind     = columns[i].orc_kind();
      if (ck->type_kind == TypeKind::STRING) {
        ck->valid_map_base   = columns[i].nulls();
        ck->column_offset    = columns[i].column_offset();
        ck->column_data_base = (ck->encoding_kind == DICTIONARY_V2)
                                 ? columns[i].host_stripe_dict(stripe_id)->dict_index
                                 : columns[i].data();
        ck->dtype_len = 1;
      } else {
        ck->valid_map_base   = columns[i].nulls();
        ck->column_offset    = columns[i].column_offset();
        ck->column_data_base = columns[i].data();
        ck->dtype_len        = columns[i].type_width();
      }
      ck->scale = columns[i].clockscale();

      // Only need to check row groups that end within the stripe
      if (ck->type_kind == TypeKind::BOOLEAN && columns[i].nullable() &&
          j + 1 != stripe_start + stripe_list[stripe_id]) {
        auto curr_cnt_in = validity_check_inputs.find(i);
        if (curr_cnt_in == validity_check_inputs.end()) {
          bool unused;
          // add new object
          std::tie(curr_cnt_in, unused) = validity_check_inputs.insert({i, {columns[i].nulls()}});
        }
        // append row group start and end to existing object
        curr_cnt_in->second.indices.push_back(ck->start_row);
        curr_cnt_in->second.indices.push_back(ck->start_row + ck->num_rows);
      }

      for (int k = 0; k < gpu::CI_NUM_STREAMS; k++) {
        const auto strm_id = strm_ids[i * gpu::CI_NUM_STREAMS + k];

        ck->strm_id[k] = strm_id;
        if (strm_id >= 0) {
          if ((k == gpu::CI_DICTIONARY) ||
              (k == gpu::CI_DATA2 && ck->encoding_kind == DICTIONARY_V2)) {
            if (j == stripe_start) {
              const int32_t dict_stride = columns[i].get_dict_stride();
              const auto stripe         = columns[i].host_stripe_dict(stripe_id);
              ck->strm_len[k]           = (k == gpu::CI_DICTIONARY)
                                  ? stripe->dict_char_count
                                  : (((stripe->num_strings + 0x1ff) >> 9) * (512 * 4 + 2));
              if (stripe_id == 0) {
                ck->streams[k] = dst_base + strm_offsets[strm_id];
              } else {
                const auto *ck_up = &chunks[stripe[-dict_stride].start_chunk * num_columns + i];
                ck->streams[k]    = ck_up->streams[k] + ck_up->strm_len[k];
              }
            } else {
              ck->strm_len[k] = 0;
              ck->streams[k]  = ck[-num_columns].streams[k];
            }
          } else if (k == gpu::CI_DATA && ck->type_kind == TypeKind::STRING &&
                     ck->encoding_kind == DIRECT_V2) {
            ck->strm_len[k] = columns[i].host_dict_chunk(j)->string_char_count;
            ck->streams[k]  = (j == 0)
                               ? dst_base + strm_offsets[strm_id]
                               : (ck[-num_columns].streams[k] + ck[-num_columns].strm_len[k]);
          } else if (k == gpu::CI_DATA && streams[strm_id].length == 0 &&
                     (ck->type_kind == DOUBLE || ck->type_kind == FLOAT)) {
            // Pass-through
            ck->strm_len[k] = ck->num_rows * ck->dtype_len;
            ck->streams[k]  = nullptr;
          } else {
            ck->strm_len[k] = streams[strm_id].length;
            ck->streams[k] =
              dst_base + str_data_size + strm_offsets[strm_id] + streams[strm_id].length * j;
          }
        } else {
          ck->strm_len[k] = 0;
          ck->streams[k]  = nullptr;
        }
      }
    }

    // Track the current stripe this rowgroup chunk belongs
    if (j + 1 == stripe_start + stripe_list[stripe_id]) {
      stripe_start = j + 1;
      stripe_id++;
    }
  }

  for (auto &cnt_in : validity_check_inputs) {
    auto const valid_counts = segmented_count_set_bits(cnt_in.second.mask, cnt_in.second.indices);
    CUDF_EXPECTS(
      std::none_of(valid_counts.cbegin(),
                   valid_counts.cend(),
                   [](auto valid_count) { return valid_count % 8; }),
      "There's currently a bug in encoding boolean columns. Suggested workaround is to convert to "
      "int8 type. Please see https://github.com/rapidsai/cudf/issues/6763 for more information.");
  }

  chunks.host_to_device(stream);
  if (!str_col_ids.empty()) {
    auto d_stripe_dict = columns[str_col_ids[0]].device_stripe_dict();
    gpu::EncodeStripeDictionaries(d_stripe_dict,
                                  chunks.device_ptr(),
                                  str_col_ids.size(),
                                  num_columns,
                                  stripe_list.size(),
                                  stream);
  }
  gpu::EncodeOrcColumnData(chunks.device_ptr(), num_columns, num_rowgroups, stream);
  stream.synchronize();

  return output;
}

std::vector<StripeInformation> writer::impl::gather_stripes(
  size_t num_columns,
  size_t num_rows,
  size_t num_index_streams,
  size_t num_data_streams,
  std::vector<uint32_t> const &stripe_list,
  hostdevice_vector<gpu::EncChunk> &chunks,
  hostdevice_vector<gpu::StripeStream> &strm_desc)
{
  std::vector<StripeInformation> stripes(stripe_list.size());
  size_t group        = 0;
  size_t stripe_start = 0;
  for (size_t s = 0; s < stripe_list.size(); s++) {
    size_t stripe_group_end = group + stripe_list[s];

    for (size_t i = 0; i < num_columns; i++) {
      const auto *ck = &chunks[group * num_columns + i];

      // Assign stream data of column data stream(s)
      for (int k = 0; k < gpu::CI_INDEX; k++) {
        const auto stream_id = ck->strm_id[k];
        if (stream_id != -1) {
          auto *ss           = &strm_desc[s * num_data_streams + stream_id - num_index_streams];
          ss->stream_size    = 0;
          ss->first_chunk_id = (group * num_columns + i);
          ss->num_chunks     = (stripe_group_end - group);
          ss->column_id      = i;
          ss->stream_type    = k;
        }
      }
    }

    group                   = stripe_group_end;
    size_t stripe_end       = std::min(group * row_index_stride_, num_rows);
    stripes[s].numberOfRows = stripe_end - stripe_start;
    stripe_start            = stripe_end;
  }

  strm_desc.host_to_device(stream);
  gpu::CompactOrcDataStreams(
    strm_desc.device_ptr(), chunks.device_ptr(), strm_desc.size(), num_columns, stream);
  strm_desc.device_to_host(stream);
  chunks.device_to_host(stream, true);

  return stripes;
}

std::vector<std::vector<uint8_t>> writer::impl::gather_statistic_blobs(
  orc_column_view const *columns,
  size_t num_columns,
  size_t num_rows,
  size_t num_rowgroups,
  std::vector<uint32_t> const &stripe_list,
  std::vector<StripeInformation> const &stripes,
  hostdevice_vector<gpu::EncChunk> &chunks)
{
  size_t num_stat_blobs = (1 + stripe_list.size()) * num_columns;
  size_t num_chunks     = chunks.size();
  std::vector<std::vector<uint8_t>> stat_blobs(num_stat_blobs);
  hostdevice_vector<stats_column_desc> stat_desc(num_columns);
  hostdevice_vector<statistics_merge_group> stat_merge(num_stat_blobs);
  rmm::device_vector<statistics_chunk> stat_chunks(num_chunks + num_stat_blobs);
  rmm::device_vector<statistics_group> stat_groups(num_chunks);

  for (size_t i = 0; i < num_columns; i++) {
    stats_column_desc *desc = &stat_desc[i];
    switch (columns[i].orc_kind()) {
      case TypeKind::BYTE: desc->stats_dtype = dtype_int8; break;
      case TypeKind::SHORT: desc->stats_dtype = dtype_int16; break;
      case TypeKind::INT: desc->stats_dtype = dtype_int32; break;
      case TypeKind::LONG: desc->stats_dtype = dtype_int64; break;
      case TypeKind::FLOAT: desc->stats_dtype = dtype_float32; break;
      case TypeKind::DOUBLE: desc->stats_dtype = dtype_float64; break;
      case TypeKind::BOOLEAN: desc->stats_dtype = dtype_bool; break;
      case TypeKind::DATE: desc->stats_dtype = dtype_int32; break;
      case TypeKind::TIMESTAMP: desc->stats_dtype = dtype_timestamp64; break;
      case TypeKind::STRING: desc->stats_dtype = dtype_string; break;
      default: desc->stats_dtype = dtype_none; break;
    }
    desc->num_rows         = columns[i].data_count();
    desc->num_values       = columns[i].data_count();
    desc->valid_map_base   = columns[i].nulls();
    desc->column_offset    = columns[i].column_offset();
    desc->column_data_base = columns[i].data();
    if (desc->stats_dtype == dtype_timestamp64) {
      // Timestamp statistics are in milliseconds
      switch (columns[i].clockscale()) {
        case 9: desc->ts_scale = 1000; break;
        case 6: desc->ts_scale = 0; break;
        case 3: desc->ts_scale = -1000; break;
        case 0: desc->ts_scale = -1000000; break;
        default: desc->ts_scale = 0; break;
      }
    } else {
      desc->ts_scale = 0;
    }
    for (size_t k = 0, c = 0; k < stripe_list.size(); k++) {
      statistics_merge_group *grp = &stat_merge[i * stripe_list.size() + k];
      grp->col                    = stat_desc.device_ptr(i);
      grp->start_chunk            = static_cast<uint32_t>(i * num_rowgroups + c);
      grp->num_chunks             = stripe_list[k];
      c += stripe_list[k];
    }
    statistics_merge_group *col_stats = &stat_merge[stripe_list.size() * num_columns + i];
    col_stats->col                    = stat_desc.device_ptr(i);
    col_stats->start_chunk            = static_cast<uint32_t>(i * stripe_list.size());
    col_stats->num_chunks             = static_cast<uint32_t>(stripe_list.size());
  }
  stat_desc.host_to_device(stream);
  stat_merge.host_to_device(stream);
  gpu::orc_init_statistics_groups(stat_groups.data().get(),
                                  stat_desc.device_ptr(),
                                  num_columns,
                                  num_rowgroups,
                                  row_index_stride_,
                                  stream);

  GatherColumnStatistics(stat_chunks.data().get(), stat_groups.data().get(), num_chunks, stream);
  MergeColumnStatistics(stat_chunks.data().get() + num_chunks,
                        stat_chunks.data().get(),
                        stat_merge.device_ptr(),
                        stripe_list.size() * num_columns,
                        stream);

  MergeColumnStatistics(stat_chunks.data().get() + num_chunks + stripe_list.size() * num_columns,
                        stat_chunks.data().get() + num_chunks,
                        stat_merge.device_ptr(stripe_list.size() * num_columns),
                        num_columns,
                        stream);
  gpu::orc_init_statistics_buffersize(
    stat_merge.device_ptr(), stat_chunks.data().get() + num_chunks, num_stat_blobs, stream);
  stat_merge.device_to_host(stream, true);

  hostdevice_vector<uint8_t> blobs(stat_merge[num_stat_blobs - 1].start_chunk +
                                   stat_merge[num_stat_blobs - 1].num_chunks);
  gpu::orc_encode_statistics(blobs.device_ptr(),
                             stat_merge.device_ptr(),
                             stat_chunks.data().get() + num_chunks,
                             num_stat_blobs,
                             stream);
  stat_merge.device_to_host(stream);
  blobs.device_to_host(stream, true);

  for (size_t i = 0; i < num_stat_blobs; i++) {
    const uint8_t *stat_begin = blobs.host_ptr(stat_merge[i].start_chunk);
    const uint8_t *stat_end   = stat_begin + stat_merge[i].num_chunks;
    stat_blobs[i].assign(stat_begin, stat_end);
  }

  return stat_blobs;
}

void writer::impl::write_index_stream(int32_t stripe_id,
                                      int32_t stream_id,
                                      orc_column_view *columns,
                                      size_t num_columns,
                                      size_t num_data_streams,
                                      size_t group,
                                      size_t groups_in_stripe,
                                      hostdevice_vector<gpu::EncChunk> const &chunks,
                                      hostdevice_vector<gpu::StripeStream> const &strm_desc,
                                      hostdevice_vector<gpu_inflate_status_s> const &comp_out,
                                      StripeInformation &stripe,
                                      std::vector<Stream> &streams,
                                      ProtobufWriter *pbw)
{
  row_group_index_info present;
  row_group_index_info data;
  row_group_index_info data2;
  auto kind = TypeKind::STRUCT;

  auto find_record = [=, &strm_desc](gpu::EncChunk const &chunk, gpu::StreamIndexType type) {
    row_group_index_info record;
    if (chunk.strm_id[type] > 0) {
      record.pos = 0;
      if (compression_kind_ != NONE) {
        const auto *ss =
          &strm_desc[stripe_id * num_data_streams + chunk.strm_id[type] - (num_columns + 1)];
        record.blk_pos   = ss->first_block;
        record.comp_pos  = 0;
        record.comp_size = ss->stream_size;
      }
    }
    return record;
  };
  auto scan_record = [=, &comp_out](gpu::EncChunk const &chunk,
                                    gpu::StreamIndexType type,
                                    row_group_index_info &record) {
    if (record.pos >= 0) {
      record.pos += chunk.strm_len[type];
      while ((record.pos >= 0) && (record.blk_pos >= 0) &&
             (static_cast<size_t>(record.pos) >= compression_blocksize_) &&
             (record.comp_pos + 3 + comp_out[record.blk_pos].bytes_written <
              static_cast<size_t>(record.comp_size))) {
        record.pos -= compression_blocksize_;
        record.comp_pos += 3 + comp_out[record.blk_pos].bytes_written;
        record.blk_pos += 1;
      }
    }
  };

  // TBD: Not sure we need an empty index stream for column 0
  if (stream_id != 0) {
    const auto &ck = chunks[stream_id - 1];
    present        = find_record(ck, gpu::CI_PRESENT);
    data           = find_record(ck, gpu::CI_DATA);
    data2          = find_record(ck, gpu::CI_DATA2);

    // Change string dictionary to int from index point of view
    kind = columns[stream_id - 1].orc_kind();
    if (kind == TypeKind::STRING && columns[stream_id - 1].orc_encoding() == DICTIONARY_V2) {
      kind = TypeKind::INT;
    }
  }

  buffer_.resize((compression_kind_ != NONE) ? 3 : 0);

  // Add row index entries
  for (size_t g = group; g < group + groups_in_stripe; g++) {
    pbw->put_row_index_entry(
      present.comp_pos, present.pos, data.comp_pos, data.pos, data2.comp_pos, data2.pos, kind);

    if (stream_id != 0) {
      const auto &ck = chunks[g * num_columns + stream_id - 1];
      scan_record(ck, gpu::CI_PRESENT, present);
      scan_record(ck, gpu::CI_DATA, data);
      scan_record(ck, gpu::CI_DATA2, data2);
    }
  }

  streams[stream_id].length = buffer_.size();
  if (compression_kind_ != NONE) {
    uint32_t uncomp_ix_len = (uint32_t)(streams[stream_id].length - 3) * 2 + 1;
    buffer_[0]             = static_cast<uint8_t>(uncomp_ix_len >> 0);
    buffer_[1]             = static_cast<uint8_t>(uncomp_ix_len >> 8);
    buffer_[2]             = static_cast<uint8_t>(uncomp_ix_len >> 16);
  }
  out_sink_->host_write(buffer_.data(), buffer_.size());
  stripe.indexLength += buffer_.size();
}

void writer::impl::write_data_stream(gpu::StripeStream const &strm_desc,
                                     gpu::EncChunk const &chunk,
                                     uint8_t const *compressed_data,
                                     uint8_t *stream_out,
                                     StripeInformation &stripe,
                                     std::vector<Stream> &streams)
{
  const auto length                                    = strm_desc.stream_size;
  streams[chunk.strm_id[strm_desc.stream_type]].length = length;
  if (length != 0) {
    const auto *stream_in = (compression_kind_ == NONE) ? chunk.streams[strm_desc.stream_type]
                                                        : (compressed_data + strm_desc.bfr_offset);
    CUDA_TRY(
      hipMemcpyAsync(stream_out, stream_in, length, hipMemcpyDeviceToHost, stream.value()));
    stream.synchronize();

    out_sink_->host_write(stream_out, length);
  }
  stripe.dataLength += length;
}

void writer::impl::add_uncompressed_block_headers(std::vector<uint8_t> &v)
{
  if (compression_kind_ != NONE) {
    size_t uncomp_len = v.size() - 3, pos = 0, block_len;
    while (uncomp_len > compression_blocksize_) {
      block_len  = compression_blocksize_ * 2 + 1;
      v[pos + 0] = static_cast<uint8_t>(block_len >> 0);
      v[pos + 1] = static_cast<uint8_t>(block_len >> 8);
      v[pos + 2] = static_cast<uint8_t>(block_len >> 16);
      pos += 3 + compression_blocksize_;
      v.insert(v.begin() + pos, 3, 0);
      uncomp_len -= compression_blocksize_;
    }
    block_len  = uncomp_len * 2 + 1;
    v[pos + 0] = static_cast<uint8_t>(block_len >> 0);
    v[pos + 1] = static_cast<uint8_t>(block_len >> 8);
    v[pos + 2] = static_cast<uint8_t>(block_len >> 16);
  }
}

writer::impl::impl(std::unique_ptr<data_sink> sink,
                   orc_writer_options const &options,
                   SingleWriteMode mode,
                   rmm::mr::device_memory_resource *mr,
                   rmm::cuda_stream_view stream)
  : compression_kind_(to_orc_compression(options.get_compression())),
    enable_statistics_(options.enable_statistics()),
    out_sink_(std::move(sink)),
    single_write_mode(mode == SingleWriteMode::YES),
    user_metadata(options.get_metadata()),
    stream(stream),
    _mr(mr)
{
  init_state();
}

writer::impl::impl(std::unique_ptr<data_sink> sink,
                   chunked_orc_writer_options const &options,
                   SingleWriteMode mode,
                   rmm::mr::device_memory_resource *mr,
                   rmm::cuda_stream_view stream)
  : compression_kind_(to_orc_compression(options.get_compression())),
    enable_statistics_(options.enable_statistics()),
    out_sink_(std::move(sink)),
    single_write_mode(mode == SingleWriteMode::YES),
    stream(stream),
    _mr(mr)
{
  if (options.get_metadata() != nullptr) {
    user_metadata_with_nullability = *options.get_metadata();
    user_metadata                  = &user_metadata_with_nullability;
  }

  init_state();
}

writer::impl::~impl() { close(); }

void writer::impl::init_state()
{
  // Write file header
  out_sink_->host_write(MAGIC, std::strlen(MAGIC));
}

void writer::impl::write(table_view const &table)
{
  CUDF_EXPECTS(not closed, "Data has already been flushed to out and closed");
  size_type num_columns = table.num_columns();
  size_type num_rows    = 0;

  // Mapping of string columns for quick look-up
  std::vector<int> str_col_ids;

  if (user_metadata_with_nullability.column_nullable.size() > 0) {
    CUDF_EXPECTS(
      user_metadata_with_nullability.column_nullable.size() == static_cast<size_t>(num_columns),
      "When passing values in user_metadata_with_nullability, data for all columns must "
      "be specified");
  }

  // Wrapper around cudf columns to attach ORC-specific type info
  std::vector<orc_column_view> orc_columns;
  orc_columns.reserve(num_columns);  // Avoids unnecessary re-allocation
  for (auto it = table.begin(); it < table.end(); ++it) {
    const auto col            = *it;
    const auto current_id     = orc_columns.size();
    const auto current_str_id = str_col_ids.size();

    num_rows = std::max<uint32_t>(num_rows, col.size());
    orc_columns.emplace_back(current_id, current_str_id, col, user_metadata, stream);
    if (orc_columns.back().is_string()) { str_col_ids.push_back(current_id); }
  }

  rmm::device_vector<uint32_t> dict_index(str_col_ids.size() * num_rows);
  rmm::device_vector<uint32_t> dict_data(str_col_ids.size() * num_rows);

  // Build per-column dictionary indices
  const auto num_rowgroups   = div_by_rowgroups<size_t>(num_rows);
  const auto num_dict_chunks = num_rowgroups * str_col_ids.size();
  hostdevice_vector<gpu::DictionaryChunk> dict(num_dict_chunks);
  if (str_col_ids.size() != 0) {
    init_dictionaries(orc_columns.data(),
                      num_rows,
                      str_col_ids,
                      dict_data.data().get(),
                      dict_index.data().get(),
                      dict);
  }

  // Decide stripe boundaries early on, based on uncompressed size
  std::vector<uint32_t> stripe_list;
  for (size_t g = 0, stripe_start = 0, stripe_size = 0; g < num_rowgroups; g++) {
    size_t rowgroup_size = 0;
    for (int i = 0; i < num_columns; i++) {
      if (orc_columns[i].is_string()) {
        const auto dt = orc_columns[i].host_dict_chunk(g);
        rowgroup_size += 1 * row_index_stride_;
        rowgroup_size += dt->string_char_count;
      } else {
        rowgroup_size += orc_columns[i].type_width() * row_index_stride_;
      }
    }

    // Apply rows per stripe limit to limit string dictionaries
    const size_t max_stripe_rows = !str_col_ids.empty() ? 1000000 : 5000000;
    if ((g > stripe_start) && (stripe_size + rowgroup_size > max_stripe_size_ ||
                               (g + 1 - stripe_start) * row_index_stride_ > max_stripe_rows)) {
      stripe_list.push_back(g - stripe_start);
      stripe_start = g;
      stripe_size  = 0;
    }
    stripe_size += rowgroup_size;
    if (g + 1 == num_rowgroups) { stripe_list.push_back(num_rowgroups - stripe_start); }
  }

  // Build stripe-level dictionaries
  const auto num_stripe_dict = stripe_list.size() * str_col_ids.size();
  hostdevice_vector<gpu::StripeDictionary> stripe_dict(num_stripe_dict);
  if (str_col_ids.size() != 0) {
    build_dictionaries(orc_columns.data(),
                       num_rows,
                       str_col_ids,
                       stripe_list,
                       dict,
                       dict_index.data().get(),
                       stripe_dict);
  }

  // Initialize streams
  std::vector<int32_t> strm_ids(num_columns * gpu::CI_NUM_STREAMS, -1);
  auto streams =
    gather_streams(orc_columns.data(), orc_columns.size(), num_rows, stripe_list, strm_ids);

  // Encode column data chunks
  const auto num_chunks = num_rowgroups * num_columns;
  hostdevice_vector<gpu::EncChunk> chunks(num_chunks);
  auto output = encode_columns(orc_columns.data(),
                               num_columns,
                               num_rows,
                               num_rowgroups,
                               str_col_ids,
                               stripe_list,
                               streams,
                               strm_ids,
                               chunks);

  // Assemble individual disparate column chunks into contiguous data streams
  const auto num_index_streams  = (num_columns + 1);
  const auto num_data_streams   = streams.size() - num_index_streams;
  const auto num_stripe_streams = stripe_list.size() * num_data_streams;
  hostdevice_vector<gpu::StripeStream> strm_desc(num_stripe_streams);
  auto stripes = gather_stripes(
    num_columns, num_rows, num_index_streams, num_data_streams, stripe_list, chunks, strm_desc);

  // Gather column statistics
  std::vector<std::vector<uint8_t>> column_stats;
  if (enable_statistics_ && num_columns > 0 && num_rows > 0) {
    column_stats = gather_statistic_blobs(
      orc_columns.data(), num_columns, num_rows, num_rowgroups, stripe_list, stripes, chunks);
  }

  // Allocate intermediate output stream buffer
  size_t compressed_bfr_size   = 0;
  size_t num_compressed_blocks = 0;
  auto stream_output           = [&]() {
    size_t max_stream_size = 0;

    for (size_t stripe_id = 0; stripe_id < stripe_list.size(); stripe_id++) {
      for (size_t i = 0; i < num_data_streams; i++) {
        gpu::StripeStream *ss = &strm_desc[stripe_id * num_data_streams + i];
        size_t stream_size    = ss->stream_size;
        if (compression_kind_ != NONE) {
          ss->first_block = num_compressed_blocks;
          ss->bfr_offset  = compressed_bfr_size;

          auto num_blocks = std::max<uint32_t>(
            (stream_size + compression_blocksize_ - 1) / compression_blocksize_, 1);
          stream_size += num_blocks * 3;
          num_compressed_blocks += num_blocks;
          compressed_bfr_size += stream_size;
        }
        max_stream_size = std::max(max_stream_size, stream_size);
      }
    }

    return pinned_buffer<uint8_t>{[](size_t size) {
                                    uint8_t *ptr = nullptr;
                                    CUDA_TRY(hipHostMalloc(&ptr, size));
                                    return ptr;
                                  }(max_stream_size),
                                  hipHostFree};
  }();

  // Compress the data streams
  rmm::device_buffer compressed_data(compressed_bfr_size, stream);
  hostdevice_vector<gpu_inflate_status_s> comp_out(num_compressed_blocks);
  hostdevice_vector<gpu_inflate_input_s> comp_in(num_compressed_blocks);
  if (compression_kind_ != NONE) {
    strm_desc.host_to_device(stream);
    gpu::CompressOrcDataStreams(static_cast<uint8_t *>(compressed_data.data()),
                                strm_desc.device_ptr(),
                                chunks.device_ptr(),
                                comp_in.device_ptr(),
                                comp_out.device_ptr(),
                                num_stripe_streams,
                                num_compressed_blocks,
                                compression_kind_,
                                compression_blocksize_,
                                stream);
    strm_desc.device_to_host(stream);
    comp_out.device_to_host(stream, true);
  }

  ProtobufWriter pbw_(&buffer_);

  // Write stripes
  size_t group = 0;
  for (size_t stripe_id = 0; stripe_id < stripes.size(); stripe_id++) {
    auto groups_in_stripe     = div_by_rowgroups(stripes[stripe_id].numberOfRows);
    stripes[stripe_id].offset = out_sink_->bytes_written();

    // Column (skippable) index streams appear at the start of the stripe
    stripes[stripe_id].indexLength = 0;
    for (size_t col_id = 0; col_id <= (size_t)num_columns; col_id++) {
      write_index_stream(stripe_id,
                         col_id,
                         orc_columns.data(),
                         num_columns,
                         num_data_streams,
                         group,
                         groups_in_stripe,
                         chunks,
                         strm_desc,
                         comp_out,
                         stripes[stripe_id],
                         streams,
                         &pbw_);
    }

    // Column data consisting one or more separate streams
    stripes[stripe_id].dataLength = 0;
    for (size_t i = 0; i < num_data_streams; i++) {
      const auto &ss = strm_desc[stripe_id * num_data_streams + i];
      const auto &ck = chunks[group * num_columns + ss.column_id];

      write_data_stream(ss,
                        ck,
                        static_cast<uint8_t *>(compressed_data.data()),
                        stream_output.get(),
                        stripes[stripe_id],
                        streams);
    }

    // Write stripefooter consisting of stream information
    StripeFooter sf;
    sf.streams = streams;
    sf.columns.resize(num_columns + 1);
    sf.columns[0].kind           = DIRECT;
    sf.columns[0].dictionarySize = 0;
    for (size_t i = 1; i < sf.columns.size(); ++i) {
      sf.columns[i].kind           = orc_columns[i - 1].orc_encoding();
      sf.columns[i].dictionarySize = (sf.columns[i].kind == DICTIONARY_V2)
                                       ? orc_columns[i - 1].host_stripe_dict(stripe_id)->num_strings
                                       : 0;
      if (orc_columns[i - 1].orc_kind() == TIMESTAMP) { sf.writerTimezone = "UTC"; }
    }
    buffer_.resize((compression_kind_ != NONE) ? 3 : 0);
    pbw_.write(sf);
    stripes[stripe_id].footerLength = buffer_.size();
    if (compression_kind_ != NONE) {
      uint32_t uncomp_sf_len = (stripes[stripe_id].footerLength - 3) * 2 + 1;
      buffer_[0]             = static_cast<uint8_t>(uncomp_sf_len >> 0);
      buffer_[1]             = static_cast<uint8_t>(uncomp_sf_len >> 8);
      buffer_[2]             = static_cast<uint8_t>(uncomp_sf_len >> 16);
    }
    out_sink_->host_write(buffer_.data(), buffer_.size());

    group += groups_in_stripe;
  }

  if (column_stats.size() != 0) {
    // File-level statistics
    // NOTE: Excluded from chunked write mode to avoid the need for merging stats across calls
    if (single_write_mode) {
      ff.statistics.resize(1 + num_columns);
      // First entry contains total number of rows
      buffer_.resize(0);
      pbw_.putb(1 * 8 + PB_TYPE_VARINT);
      pbw_.put_uint(num_rows);
      ff.statistics[0] = std::move(buffer_);
      for (int i = 0; i < num_columns; i++) {
        size_t idx = stripe_list.size() * num_columns + i;
        if (idx < column_stats.size()) { ff.statistics[1 + i] = std::move(column_stats[idx]); }
      }
    }
    // Stripe-level statistics
    size_t first_stripe = md.stripeStats.size();
    md.stripeStats.resize(first_stripe + stripe_list.size());
    for (size_t stripe_id = 0; stripe_id < stripe_list.size(); stripe_id++) {
      md.stripeStats[first_stripe + stripe_id].colStats.resize(1 + num_columns);
      buffer_.resize(0);
      pbw_.putb(1 * 8 + PB_TYPE_VARINT);
      pbw_.put_uint(stripes[stripe_id].numberOfRows);
      md.stripeStats[first_stripe + stripe_id].colStats[0] = std::move(buffer_);
      for (int i = 0; i < num_columns; i++) {
        size_t idx = stripe_list.size() * i + stripe_id;
        if (idx < column_stats.size()) {
          md.stripeStats[first_stripe + stripe_id].colStats[1 + i] = std::move(column_stats[idx]);
        }
      }
    }
  }
  if (ff.headerLength == 0) {
    // First call
    ff.headerLength   = std::strlen(MAGIC);
    ff.rowIndexStride = row_index_stride_;
    ff.types.resize(1 + num_columns);
    ff.types[0].kind = STRUCT;
    ff.types[0].subtypes.resize(num_columns);
    ff.types[0].fieldNames.resize(num_columns);
    for (int i = 0; i < num_columns; ++i) {
      ff.types[1 + i].kind      = orc_columns[i].orc_kind();
      ff.types[0].subtypes[i]   = 1 + i;
      ff.types[0].fieldNames[i] = orc_columns[i].orc_name();
    }
  } else {
    // verify the user isn't passing mismatched tables
    CUDF_EXPECTS(ff.types.size() == 1 + orc_columns.size(),
                 "Mismatch in table structure between multiple calls to write");
    for (auto i = 0; i < num_columns; i++) {
      CUDF_EXPECTS(ff.types[1 + i].kind == orc_columns[i].orc_kind(),
                   "Mismatch in column types between multiple calls to write");
    }
  }
  ff.stripes.insert(ff.stripes.end(),
                    std::make_move_iterator(stripes.begin()),
                    std::make_move_iterator(stripes.end()));
  ff.numberOfRows += num_rows;
}

void writer::impl::close()
{
  if (closed) { return; }
  closed = true;
  ProtobufWriter pbw_(&buffer_);
  PostScript ps;

  ff.contentLength = out_sink_->bytes_written();
  if (user_metadata) {
    for (auto it = user_metadata->user_data.begin(); it != user_metadata->user_data.end(); it++) {
      ff.metadata.push_back({it->first, it->second});
    }
  }
  // Write statistics metadata
  if (md.stripeStats.size() != 0) {
    buffer_.resize((compression_kind_ != NONE) ? 3 : 0);
    pbw_.write(md);
    add_uncompressed_block_headers(buffer_);
    ps.metadataLength = buffer_.size();
    out_sink_->host_write(buffer_.data(), buffer_.size());
  } else {
    ps.metadataLength = 0;
  }
  buffer_.resize((compression_kind_ != NONE) ? 3 : 0);
  pbw_.write(ff);
  add_uncompressed_block_headers(buffer_);

  // Write postscript metadata
  ps.footerLength         = buffer_.size();
  ps.compression          = compression_kind_;
  ps.compressionBlockSize = compression_blocksize_;
  ps.version              = {0, 12};
  ps.magic                = MAGIC;
  const auto ps_length    = static_cast<uint8_t>(pbw_.write(ps));
  buffer_.push_back(ps_length);
  out_sink_->host_write(buffer_.data(), buffer_.size());
  out_sink_->flush();
}

// Forward to implementation
writer::writer(std::unique_ptr<data_sink> sink,
               orc_writer_options const &options,
               SingleWriteMode mode,
               rmm::mr::device_memory_resource *mr,
               rmm::cuda_stream_view stream)
  : _impl(std::make_unique<impl>(std::move(sink), options, mode, mr, stream))
{
}

// Forward to implementation
writer::writer(std::unique_ptr<data_sink> sink,
               chunked_orc_writer_options const &options,
               SingleWriteMode mode,
               rmm::mr::device_memory_resource *mr,
               rmm::cuda_stream_view stream)
  : _impl(std::make_unique<impl>(std::move(sink), options, mode, mr, stream))
{
}

// Destructor within this translation unit
writer::~writer() = default;

// Forward to implementation
void writer::write(table_view const &table) { _impl->write(table); }

// Forward to implementation
void writer::close() { _impl->close(); }

}  // namespace orc
}  // namespace detail
}  // namespace io
}  // namespace cudf
