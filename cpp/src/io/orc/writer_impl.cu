#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019-2021, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

/**
 * @file writer_impl.cu
 * @brief cuDF-IO ORC writer class implementation
 */

#include "writer_impl.hpp"

#include <io/statistics/column_statistics.cuh>
#include <io/utilities/column_utils.cuh>

#include <cudf/detail/iterator.cuh>
#include <cudf/detail/utilities/cuda.cuh>
#include <cudf/detail/utilities/vector_factories.hpp>
#include <cudf/null_mask.hpp>
#include <cudf/strings/strings_column_view.hpp>
#include <cudf/utilities/bit.hpp>
#include <cudf/utilities/span.hpp>

#include <rmm/cuda_stream_view.hpp>
#include <rmm/device_buffer.hpp>
#include <rmm/device_uvector.hpp>

#include <algorithm>
#include <cstring>
#include <numeric>
#include <utility>

namespace cudf {
namespace io {
namespace detail {
namespace orc {
using namespace cudf::io::orc;
using namespace cudf::io;

struct row_group_index_info {
  int32_t pos       = -1;  // Position
  int32_t blk_pos   = -1;  // Block Position
  int32_t comp_pos  = -1;  // Compressed Position
  int32_t comp_size = -1;  // Compressed size
};

namespace {
/**
 * @brief Helper for pinned host memory
 */
template <typename T>
using pinned_buffer = std::unique_ptr<T, decltype(&hipHostFree)>;

/**
 * @brief Function that translates GDF compression to ORC compression
 */
orc::CompressionKind to_orc_compression(compression_type compression)
{
  switch (compression) {
    case compression_type::AUTO:
    case compression_type::SNAPPY: return orc::CompressionKind::SNAPPY;
    case compression_type::NONE: return orc::CompressionKind::NONE;
    default: CUDF_EXPECTS(false, "Unsupported compression type"); return orc::CompressionKind::NONE;
  }
}

/**
 * @brief Function that translates GDF dtype to ORC datatype
 */
constexpr orc::TypeKind to_orc_type(cudf::type_id id)
{
  switch (id) {
    case cudf::type_id::INT8: return TypeKind::BYTE;
    case cudf::type_id::INT16: return TypeKind::SHORT;
    case cudf::type_id::INT32: return TypeKind::INT;
    case cudf::type_id::INT64: return TypeKind::LONG;
    case cudf::type_id::FLOAT32: return TypeKind::FLOAT;
    case cudf::type_id::FLOAT64: return TypeKind::DOUBLE;
    case cudf::type_id::BOOL8: return TypeKind::BOOLEAN;
    case cudf::type_id::TIMESTAMP_DAYS: return TypeKind::DATE;
    case cudf::type_id::TIMESTAMP_SECONDS:
    case cudf::type_id::TIMESTAMP_MICROSECONDS:
    case cudf::type_id::TIMESTAMP_MILLISECONDS:
    case cudf::type_id::TIMESTAMP_NANOSECONDS: return TypeKind::TIMESTAMP;
    case cudf::type_id::STRING: return TypeKind::STRING;
    case cudf::type_id::DECIMAL32:
    case cudf::type_id::DECIMAL64: return TypeKind::DECIMAL;
    case cudf::type_id::LIST: return TypeKind::LIST;
    default: return TypeKind::INVALID_TYPE_KIND;
  }
}

/**
 * @brief Translates time unit to nanoscale multiple.
 */
constexpr int32_t to_clockscale(cudf::type_id timestamp_id)
{
  switch (timestamp_id) {
    case cudf::type_id::TIMESTAMP_SECONDS: return 9;
    case cudf::type_id::TIMESTAMP_MILLISECONDS: return 6;
    case cudf::type_id::TIMESTAMP_MICROSECONDS: return 3;
    case cudf::type_id::TIMESTAMP_NANOSECONDS:
    default: return 0;
  }
}

/**
 * @brief Returns the precision of the given decimal type.
 */
constexpr auto orc_precision(cudf::type_id decimal_id)
{
  switch (decimal_id) {
    case cudf::type_id::DECIMAL32: return 9;
    case cudf::type_id::DECIMAL64: return 18;
    default: return 0;
  }
}

}  // namespace

/**
 * @brief Helper class that adds ORC-specific column info
 */
class orc_column_view {
 public:
  /**
   * @brief Constructor that extracts out the string position + length pairs
   * for building dictionaries for string columns
   */
  explicit orc_column_view(uint32_t index,
                           std::optional<uint32_t> str_idx,
                           std::optional<uint32_t> index_in_table,
                           column_view const& col,
                           const table_metadata* metadata)
    : cudf_column{col},
      _index{index},
      _str_idx{str_idx},
      _is_child{!index_in_table.has_value()},
      _type_width{cudf::is_fixed_width(col.type()) ? cudf::size_of(col.type()) : 0},
      _scale{(to_orc_type(col.type().id()) == TypeKind::DECIMAL) ? -col.type().scale()
                                                                 : to_clockscale(col.type().id())},
      _precision{orc_precision(col.type().id())},
      _type_kind{to_orc_type(col.type().id())}
  {
    // Don't assign names to child columns
    if (index_in_table.has_value()) {
      if (metadata != nullptr && *index_in_table < metadata->column_names.size()) {
        _name = metadata->column_names[*index_in_table];
      } else {
        // Generating default name if name isn't present in metadata
        _name = "_col" + std::to_string(*index_in_table);
      }
    }
  }

  auto is_string() const noexcept { return cudf_column.type().id() == type_id::STRING; }
  void set_dict_stride(size_t stride) noexcept { dict_stride = stride; }
  auto get_dict_stride() const noexcept { return dict_stride; }

  /**
   * @brief Function that associates an existing dictionary chunk allocation
   */
  void attach_dict_chunk(gpu::DictionaryChunk const* host_dict,
                         gpu::DictionaryChunk const* dev_dict)
  {
    dict   = host_dict;
    d_dict = dev_dict;
  }
  auto host_dict_chunk(size_t rowgroup) const
  {
    CUDF_EXPECTS(is_string(), "Dictionary chunks are only present in string columns.");
    return &dict[rowgroup * dict_stride + *_str_idx];
  }
  auto device_dict_chunk() const { return d_dict; }

  auto const& decimal_offsets() const { return d_decimal_offsets; }
  void attach_decimal_offsets(uint32_t* sizes_ptr) { d_decimal_offsets = sizes_ptr; }

  /**
   * @brief Function that associates an existing stripe dictionary allocation
   */
  void attach_stripe_dict(gpu::StripeDictionary* host_stripe_dict,
                          gpu::StripeDictionary* dev_stripe_dict)
  {
    stripe_dict   = host_stripe_dict;
    d_stripe_dict = dev_stripe_dict;
  }
  auto host_stripe_dict(size_t stripe) const
  {
    CUDF_EXPECTS(is_string(), "Stripe dictionary is only present in string columns.");
    return &stripe_dict[stripe * dict_stride + *_str_idx];
  }
  auto device_stripe_dict() const { return d_stripe_dict; }

  // Index in the table
  auto index() const noexcept { return _index; }
  // Id in the ORC file
  auto id() const noexcept { return _index + 1; }
  auto is_child() const noexcept { return _is_child; }
  auto type_width() const noexcept { return _type_width; }
  auto size() const noexcept { return cudf_column.size(); }
  auto null_count() const noexcept { return cudf_column.null_count(); }
  auto null_mask() const noexcept { return cudf_column.null_mask(); }
  bool nullable() const noexcept { return null_mask() != nullptr; }

  auto scale() const noexcept { return _scale; }
  auto precision() const noexcept { return _precision; }

  void set_orc_encoding(ColumnEncodingKind e) noexcept { _encoding_kind = e; }
  auto orc_kind() const noexcept { return _type_kind; }
  auto orc_encoding() const noexcept { return _encoding_kind; }
  auto orc_name() const noexcept { return _name; }

 private:
  column_view cudf_column;

  // Identifier within the set of columns
  uint32_t _index = 0;
  // Identifier within the set of string columns
  std::optional<uint32_t> _str_idx;
  bool _is_child = false;

  size_t _type_width = 0;
  int32_t _scale     = 0;
  int32_t _precision = 0;

  // ORC-related members
  std::string _name{};
  TypeKind _type_kind;
  ColumnEncodingKind _encoding_kind;

  // String dictionary-related members
  size_t dict_stride                         = 0;
  gpu::DictionaryChunk const* dict           = nullptr;
  gpu::StripeDictionary const* stripe_dict   = nullptr;
  gpu::DictionaryChunk const* d_dict         = nullptr;
  gpu::StripeDictionary const* d_stripe_dict = nullptr;

  // Offsets for encoded decimal elements. Used to enable direct writing of encoded decimal elements
  // into the output stream.
  uint32_t* d_decimal_offsets = nullptr;
};

size_type orc_table_view::num_rows() const noexcept
{
  return columns.empty() ? 0 : columns.front().size();
}

/**
 * @brief Gathers stripe information.
 *
 * @param columns List of columns
 * @param rowgroup_bounds Ranges of rows in each rowgroup [rowgroup][column]
 * @param max_stripe_bytes Maximum size of each stripe, in bytes
 * @return List of stripe descriptors
 */
file_segmentation calculate_segmentation(host_span<orc_column_view const> columns,
                                         hostdevice_2dvector<rowgroup_rows>&& rowgroup_bounds,
                                         uint32_t max_stripe_bytes)
{
  auto const is_any_column_string =
    std::any_of(columns.begin(), columns.end(), [](auto const& col) { return col.is_string(); });
  // Apply rows per stripe limit to limit string dictionaries
  size_t const max_stripe_rows = is_any_column_string ? 1000000 : 5000000;

  std::vector<stripe_rowgroups> infos;
  auto const num_rowgroups = rowgroup_bounds.size().first;
  size_t stripe_start      = 0;
  size_t stripe_bytes      = 0;
  size_t stripe_rows       = 0;
  for (size_t rg_idx = 0; rg_idx < num_rowgroups; ++rg_idx) {
    auto const rowgroup_total_bytes =
      std::accumulate(columns.begin(), columns.end(), 0ul, [&](size_t total_size, auto const& col) {
        auto const rows = rowgroup_bounds[rg_idx][col.index()].size();
        if (col.is_string()) {
          const auto dt = col.host_dict_chunk(rg_idx);
          return total_size + rows + dt->string_char_count;
        } else {
          return total_size + col.type_width() * rows;
        }
      });

    auto const rowgroup_rows_max =
      std::max_element(rowgroup_bounds[rg_idx].begin(),
                       rowgroup_bounds[rg_idx].end(),
                       [](auto& l, auto& r) { return l.size() < r.size(); })
        ->size();
    // Check if adding the current rowgroup to the stripe will make the stripe too large or long
    if ((rg_idx > stripe_start) && (stripe_bytes + rowgroup_total_bytes > max_stripe_bytes ||
                                    stripe_rows + rowgroup_rows_max > max_stripe_rows)) {
      infos.emplace_back(infos.size(), stripe_start, rg_idx - stripe_start);
      stripe_start = rg_idx;
      stripe_bytes = 0;
      stripe_rows  = 0;
    }

    stripe_bytes += rowgroup_total_bytes;
    stripe_rows += rowgroup_rows_max;
    if (rg_idx + 1 == num_rowgroups) {
      infos.emplace_back(infos.size(), stripe_start, num_rowgroups - stripe_start);
    }
  }

  return {std::move(rowgroup_bounds), std::move(infos)};
}

/**
 * @brief Builds up column dictionaries indices
 *
 * @param orc_table Non-owning view of a cuDF table w/ ORC-related info
 * @param rowgroup_bounds Ranges of rows in each rowgroup [rowgroup][column]
 * @param dict_data Dictionary data memory
 * @param dict_index Dictionary index memory
 * @param dict List of dictionary chunks
 * @param stream CUDA stream used for device memory operations and kernel launches
 */
void init_dictionaries(orc_table_view& orc_table,
                       device_2dspan<rowgroup_rows const> rowgroup_bounds,
                       device_span<device_span<uint32_t>> dict_data,
                       device_span<device_span<uint32_t>> dict_index,
                       hostdevice_vector<gpu::DictionaryChunk>* dict,
                       rmm::cuda_stream_view stream)
{
  // Setup per-rowgroup dictionary indexes for each dictionary-aware column
  for (auto col_idx : orc_table.string_column_indices) {
    auto& str_column = orc_table.column(col_idx);
    str_column.set_dict_stride(orc_table.num_string_columns());
    str_column.attach_dict_chunk(dict->host_ptr(), dict->device_ptr());
  }

  // Allocate temporary memory for dictionary indices
  std::vector<rmm::device_uvector<uint32_t>> dict_indices;
  dict_indices.reserve(orc_table.num_string_columns());
  std::transform(orc_table.string_column_indices.cbegin(),
                 orc_table.string_column_indices.cend(),
                 std::back_inserter(dict_indices),
                 [&](auto& col_idx) {
                   auto& str_column = orc_table.column(col_idx);
                   return cudf::detail::make_zeroed_device_uvector_async<uint32_t>(
                     str_column.size(), stream);
                 });

  // Create views of the temporary buffers in device memory
  std::vector<device_span<uint32_t>> dict_indices_views;
  dict_indices_views.reserve(dict_indices.size());
  std::transform(
    dict_indices.begin(), dict_indices.end(), std::back_inserter(dict_indices_views), [](auto& di) {
      return device_span<uint32_t>{di};
    });
  auto d_dict_indices_views = cudf::detail::make_device_uvector_async(dict_indices_views, stream);

  gpu::InitDictionaryIndices(orc_table.d_columns,
                             dict->device_ptr(),
                             dict_data,
                             dict_index,
                             d_dict_indices_views,
                             rowgroup_bounds,
                             orc_table.d_string_column_indices,
                             stream);
  dict->device_to_host(stream, true);
}

void writer::impl::build_dictionaries(orc_table_view& orc_table,
                                      host_span<stripe_rowgroups const> stripe_bounds,
                                      hostdevice_vector<gpu::DictionaryChunk> const& dict,
                                      host_span<rmm::device_uvector<uint32_t>> dict_index,
                                      host_span<bool const> dictionary_enabled,
                                      hostdevice_vector<gpu::StripeDictionary>& stripe_dict)
{
  const auto num_rowgroups = dict.size() / orc_table.num_string_columns();

  for (size_t dict_idx = 0; dict_idx < orc_table.num_string_columns(); ++dict_idx) {
    auto& str_column = orc_table.string_column(dict_idx);
    str_column.attach_stripe_dict(stripe_dict.host_ptr(), stripe_dict.device_ptr());

    for (auto const& stripe : stripe_bounds) {
      auto& sd           = stripe_dict[stripe.id * orc_table.num_string_columns() + dict_idx];
      sd.dict_data       = str_column.host_dict_chunk(stripe.first)->dict_data;
      sd.dict_index      = dict_index[dict_idx].data();  // Indexed by abs row
      sd.column_id       = orc_table.string_column_indices[dict_idx];
      sd.start_chunk     = stripe.first;
      sd.num_chunks      = stripe.size;
      sd.dict_char_count = 0;
      sd.num_strings =
        std::accumulate(stripe.cbegin(), stripe.cend(), 0, [&](auto dt_str_cnt, auto rg_idx) {
          const auto& dt = dict[rg_idx * orc_table.num_string_columns() + dict_idx];
          return dt_str_cnt + dt.num_dict_strings;
        });
      sd.leaf_column = dict[dict_idx].leaf_column;
    }

    if (enable_dictionary_) {
      struct string_column_cost {
        size_t direct     = 0;
        size_t dictionary = 0;
      };
      auto const col_cost =
        std::accumulate(stripe_bounds.front().cbegin(),
                        stripe_bounds.back().cend(),
                        string_column_cost{},
                        [&](auto cost, auto rg_idx) -> string_column_cost {
                          const auto& dt = dict[rg_idx * orc_table.num_string_columns() + dict_idx];
                          return {cost.direct + dt.string_char_count,
                                  cost.dictionary + dt.dict_char_count + dt.num_dict_strings};
                        });
      // Disable dictionary if it does not reduce the output size
      if (!dictionary_enabled[orc_table.string_column(dict_idx).index()] ||
          col_cost.dictionary >= col_cost.direct) {
        for (auto const& stripe : stripe_bounds) {
          stripe_dict[stripe.id * orc_table.num_string_columns() + dict_idx].dict_data = nullptr;
        }
      }
    }
  }

  stripe_dict.host_to_device(stream);
  gpu::BuildStripeDictionaries(stripe_dict.device_ptr(),
                               stripe_dict.host_ptr(),
                               dict.device_ptr(),
                               stripe_bounds.size(),
                               num_rowgroups,
                               orc_table.string_column_indices.size(),
                               stream);
  stripe_dict.device_to_host(stream, true);
}

constexpr size_t RLE_stream_size(TypeKind kind, size_t count)
{
  using cudf::util::div_rounding_up_unsafe;
  switch (kind) {
    case TypeKind::BOOLEAN: return div_rounding_up_unsafe(count, 1024) * (128 + 1);
    case TypeKind::BYTE: return div_rounding_up_unsafe(count, 128) * (128 + 1);
    case TypeKind::SHORT: return div_rounding_up_unsafe(count, 512) * (512 * 2 + 2);
    case TypeKind::FLOAT:
    case TypeKind::INT:
    case TypeKind::DATE: return div_rounding_up_unsafe(count, 512) * (512 * 4 + 2);
    case TypeKind::LONG:
    case TypeKind::DOUBLE: return div_rounding_up_unsafe(count, 512) * (512 * 8 + 2);
    default: CUDF_FAIL("Unsupported ORC type for RLE stream size");
  }
}

orc_streams writer::impl::create_streams(host_span<orc_column_view> columns,
                                         file_segmentation const& segmentation,
                                         std::map<uint32_t, size_t> const& decimal_column_sizes)
{
  // 'column 0' row index stream
  std::vector<Stream> streams{{ROW_INDEX, 0}};  // TODO: Separate index and data streams?
  // First n + 1 streams are row index streams
  streams.reserve(columns.size() + 1);
  std::transform(columns.begin(), columns.end(), std::back_inserter(streams), [](auto const& col) {
    return Stream{ROW_INDEX, col.id()};
  });

  std::vector<int32_t> ids(columns.size() * gpu::CI_NUM_STREAMS, -1);

  for (auto& column : columns) {
    auto const is_nullable = [&]() -> bool {
      if (single_write_mode) {
        return column.nullable();
      } else {
        if (user_metadata_with_nullability.column_nullable.empty()) return true;
        CUDF_EXPECTS(user_metadata_with_nullability.column_nullable.size() > column.index(),
                     "When passing values in user_metadata_with_nullability, data for all columns "
                     "must be specified");
        return user_metadata_with_nullability.column_nullable[column.index()];
      }
    }();

    auto RLE_column_size = [&](TypeKind type_kind) {
      return std::accumulate(
        thrust::make_counting_iterator(0ul),
        thrust::make_counting_iterator(segmentation.num_rowgroups()),
        0ul,
        [&](auto data_size, auto rg_idx) {
          return data_size +
                 RLE_stream_size(type_kind, segmentation.rowgroups[rg_idx][column.index()].size());
        });
    };

    auto const kind = column.orc_kind();

    auto add_stream =
      [&](gpu::StreamIndexType index_type, StreamKind kind, TypeKind type_kind, size_t size) {
        const auto base        = column.index() * gpu::CI_NUM_STREAMS;
        ids[base + index_type] = streams.size();
        streams.push_back(orc::Stream{kind, column.id(), size, type_kind});
      };

    auto add_RLE_stream = [&](
                            gpu::StreamIndexType index_type, StreamKind kind, TypeKind type_kind) {
      add_stream(index_type, kind, type_kind, RLE_column_size(type_kind));
    };

    if (is_nullable) { add_RLE_stream(gpu::CI_PRESENT, PRESENT, TypeKind::BOOLEAN); }
    switch (kind) {
      case TypeKind::BOOLEAN:
      case TypeKind::BYTE:
        add_RLE_stream(gpu::CI_DATA, DATA, kind);
        column.set_orc_encoding(DIRECT);
        break;
      case TypeKind::SHORT:
      case TypeKind::INT:
      case TypeKind::LONG:
      case TypeKind::DATE:
        add_RLE_stream(gpu::CI_DATA, DATA, kind);
        column.set_orc_encoding(DIRECT_V2);
        break;
      case TypeKind::FLOAT:
      case TypeKind::DOUBLE:
        // Pass through if no nulls (no RLE encoding for floating point)
        add_stream(
          gpu::CI_DATA, DATA, kind, (column.null_count() != 0) ? RLE_column_size(kind) : 0);
        column.set_orc_encoding(DIRECT);
        break;
      case TypeKind::STRING: {
        bool enable_dict           = enable_dictionary_;
        size_t dict_data_size      = 0;
        size_t dict_strings        = 0;
        size_t dict_lengths_div512 = 0;
        for (auto const& stripe : segmentation.stripes) {
          const auto sd = column.host_stripe_dict(stripe.id);
          enable_dict   = (enable_dict && sd->dict_data != nullptr);
          if (enable_dict) {
            dict_strings += sd->num_strings;
            dict_lengths_div512 += (sd->num_strings + 0x1ff) >> 9;
            dict_data_size += sd->dict_char_count;
          }
        }

        auto const direct_data_size =
          std::accumulate(segmentation.stripes.front().cbegin(),
                          segmentation.stripes.back().cend(),
                          size_t{0},
                          [&](auto data_size, auto rg_idx) {
                            return data_size + column.host_dict_chunk(rg_idx)->string_char_count;
                          });
        if (enable_dict) {
          uint32_t dict_bits = 0;
          for (dict_bits = 1; dict_bits < 32; dict_bits <<= 1) {
            if (dict_strings <= (1ull << dict_bits)) break;
          }
          const auto valid_count = column.size() - column.null_count();
          dict_data_size += (dict_bits * valid_count + 7) >> 3;
        }

        // Decide between direct or dictionary encoding
        if (enable_dict && dict_data_size < direct_data_size) {
          add_RLE_stream(gpu::CI_DATA, DATA, TypeKind::INT);
          add_stream(gpu::CI_DATA2, LENGTH, TypeKind::INT, dict_lengths_div512 * (512 * 4 + 2));
          add_stream(
            gpu::CI_DICTIONARY, DICTIONARY_DATA, TypeKind::CHAR, std::max(dict_data_size, 1ul));
          column.set_orc_encoding(DICTIONARY_V2);
        } else {
          add_stream(gpu::CI_DATA, DATA, TypeKind::CHAR, std::max<size_t>(direct_data_size, 1));
          add_RLE_stream(gpu::CI_DATA2, LENGTH, TypeKind::INT);
          column.set_orc_encoding(DIRECT_V2);
        }
        break;
      }
      case TypeKind::TIMESTAMP:
        add_RLE_stream(gpu::CI_DATA, DATA, TypeKind::INT);
        add_RLE_stream(gpu::CI_DATA2, SECONDARY, TypeKind::INT);
        column.set_orc_encoding(DIRECT_V2);
        break;
      case TypeKind::DECIMAL:
        // varint values (NO RLE)
        // data_stream_size = decimal_column_sizes.at(column.index());
        add_stream(gpu::CI_DATA, DATA, TypeKind::DECIMAL, decimal_column_sizes.at(column.index()));
        // scale stream TODO: compute exact size since all elems are equal
        add_RLE_stream(gpu::CI_DATA2, SECONDARY, TypeKind::INT);
        column.set_orc_encoding(DIRECT_V2);
        break;
      case TypeKind::LIST:
        // no data stream, only lengths
        add_RLE_stream(gpu::CI_DATA2, LENGTH, TypeKind::INT);
        column.set_orc_encoding(DIRECT_V2);
        break;
      default: CUDF_FAIL("Unsupported ORC type kind");
    }
  }
  return {std::move(streams), std::move(ids)};
}

orc_streams::orc_stream_offsets orc_streams::compute_offsets(
  host_span<orc_column_view const> columns, size_t num_rowgroups) const
{
  std::vector<size_t> strm_offsets(streams.size());
  size_t non_rle_data_size = 0;
  size_t rle_data_size     = 0;
  for (size_t i = 0; i < streams.size(); ++i) {
    const auto& stream = streams[i];

    auto const is_rle_data = [&]() {
      // First stream is an index stream, don't check types, etc.
      if (!stream.column_index().has_value()) return true;

      auto const& column = columns[stream.column_index().value()];
      // Dictionary encoded string column - dictionary characters or
      // directly encoded string - column characters
      if (column.orc_kind() == TypeKind::STRING &&
          ((stream.kind == DICTIONARY_DATA && column.orc_encoding() == DICTIONARY_V2) ||
           (stream.kind == DATA && column.orc_encoding() == DIRECT_V2)))
        return false;
      // Decimal data
      if (column.orc_kind() == TypeKind::DECIMAL && stream.kind == DATA) return false;

      // Everything else uses RLE
      return true;
    }();
    // non-RLE and RLE streams are separated in the buffer that stores encoded data
    // The computed offsets do not take the streams of the other type into account
    if (is_rle_data) {
      strm_offsets[i] = rle_data_size;
      rle_data_size += (stream.length + 7) & ~7;
    } else {
      strm_offsets[i] = non_rle_data_size;
      non_rle_data_size += stream.length;
    }
  }
  non_rle_data_size = (non_rle_data_size + 7) & ~7;

  return {std::move(strm_offsets), non_rle_data_size, rle_data_size};
}

struct segmented_valid_cnt_input {
  bitmask_type const* mask;
  std::vector<size_type> indices;
};

encoded_data writer::impl::encode_columns(orc_table_view const& orc_table,
                                          string_dictionaries&& dictionaries,
                                          encoder_decimal_info&& dec_chunk_sizes,
                                          file_segmentation const& segmentation,
                                          orc_streams const& streams)
{
  auto const num_columns = orc_table.num_columns();
  hostdevice_2dvector<gpu::EncChunk> chunks(num_columns, segmentation.num_rowgroups(), stream);
  auto const stream_offsets =
    streams.compute_offsets(orc_table.columns, segmentation.num_rowgroups());
  rmm::device_uvector<uint8_t> encoded_data(stream_offsets.data_size(), stream);

  // Initialize column chunks' descriptions
  std::map<size_type, segmented_valid_cnt_input> validity_check_inputs;

  for (auto const& column : orc_table.columns) {
    for (auto const& stripe : segmentation.stripes) {
      for (auto rg_idx_it = stripe.cbegin(); rg_idx_it < stripe.cend(); ++rg_idx_it) {
        auto const rg_idx = *rg_idx_it;
        auto& ck          = chunks[column.index()][rg_idx];

        ck.start_row     = segmentation.rowgroups[rg_idx][column.index()].begin;
        ck.num_rows      = segmentation.rowgroups[rg_idx][column.index()].size();
        ck.encoding_kind = column.orc_encoding();
        ck.type_kind     = column.orc_kind();
        if (ck.type_kind == TypeKind::STRING) {
          ck.dict_index = (ck.encoding_kind == DICTIONARY_V2)
                            ? column.host_stripe_dict(stripe.id)->dict_index
                            : nullptr;
          ck.dtype_len  = 1;
        } else {
          ck.dtype_len = column.type_width();
        }
        ck.scale = column.scale();
        if (ck.type_kind == TypeKind::DECIMAL) {
          ck.decimal_offsets = device_span<uint32_t>{column.decimal_offsets(), ck.num_rows};
        }
      }
    }
  }

  auto validity_check_indices = [&](size_t col_idx) {
    std::vector<size_type> indices;
    for (auto const& stripe : segmentation.stripes) {
      for (auto rg_idx_it = stripe.cbegin(); rg_idx_it < stripe.cend() - 1; ++rg_idx_it) {
        auto const& chunk = chunks[col_idx][*rg_idx_it];
        indices.push_back(chunk.start_row);
        indices.push_back(chunk.start_row + chunk.num_rows);
      }
    }
    return indices;
  };
  for (auto const& column : orc_table.columns) {
    if (column.orc_kind() == TypeKind::BOOLEAN && column.nullable()) {
      validity_check_inputs[column.index()] = {column.null_mask(),
                                               validity_check_indices(column.index())};
    }
  }
  for (auto& cnt_in : validity_check_inputs) {
    auto const valid_counts = segmented_count_set_bits(cnt_in.second.mask, cnt_in.second.indices);
    CUDF_EXPECTS(
      std::none_of(valid_counts.cbegin(),
                   valid_counts.cend(),
                   [](auto valid_count) { return valid_count % 8; }),
      "There's currently a bug in encoding boolean columns. Suggested workaround is to convert "
      "to int8 type."
      " Please see https://github.com/rapidsai/cudf/issues/6763 for more information.");
  }

  hostdevice_2dvector<gpu::encoder_chunk_streams> chunk_streams(
    num_columns, segmentation.num_rowgroups(), stream);
  for (size_t col_idx = 0; col_idx < num_columns; col_idx++) {
    auto const& column = orc_table.column(col_idx);
    auto col_streams   = chunk_streams[col_idx];
    for (auto const& stripe : segmentation.stripes) {
      for (auto rg_idx_it = stripe.cbegin(); rg_idx_it < stripe.cend(); ++rg_idx_it) {
        auto const rg_idx = *rg_idx_it;
        auto const& ck    = chunks[col_idx][rg_idx];
        auto& strm        = col_streams[rg_idx];

        for (int strm_type = 0; strm_type < gpu::CI_NUM_STREAMS; ++strm_type) {
          auto const strm_id = streams.id(col_idx * gpu::CI_NUM_STREAMS + strm_type);

          strm.ids[strm_type] = strm_id;
          if (strm_id >= 0) {
            if ((strm_type == gpu::CI_DICTIONARY) ||
                (strm_type == gpu::CI_DATA2 && ck.encoding_kind == DICTIONARY_V2)) {
              if (rg_idx_it == stripe.cbegin()) {
                const int32_t dict_stride = column.get_dict_stride();
                const auto stripe_dict    = column.host_stripe_dict(stripe.id);
                strm.lengths[strm_type] =
                  (strm_type == gpu::CI_DICTIONARY)
                    ? stripe_dict->dict_char_count
                    : (((stripe_dict->num_strings + 0x1ff) >> 9) * (512 * 4 + 2));
                if (stripe.id == 0) {
                  strm.data_ptrs[strm_type] = encoded_data.data() + stream_offsets.offsets[strm_id];
                  // Dictionary lengths are encoded as RLE, which are all stored after non-RLE data:
                  // include non-RLE data size in the offset only in that case
                  if (strm_type == gpu::CI_DATA2 && ck.encoding_kind == DICTIONARY_V2)
                    strm.data_ptrs[strm_type] += stream_offsets.non_rle_data_size;
                } else {
                  auto const& strm_up = col_streams[stripe_dict[-dict_stride].start_chunk];
                  strm.data_ptrs[strm_type] =
                    strm_up.data_ptrs[strm_type] + strm_up.lengths[strm_type];
                }
              } else {
                strm.lengths[strm_type]   = 0;
                strm.data_ptrs[strm_type] = col_streams[rg_idx - 1].data_ptrs[strm_type];
              }
            } else if (strm_type == gpu::CI_DATA && ck.type_kind == TypeKind::STRING &&
                       ck.encoding_kind == DIRECT_V2) {
              strm.lengths[strm_type]   = column.host_dict_chunk(rg_idx)->string_char_count;
              strm.data_ptrs[strm_type] = (rg_idx == 0)
                                            ? encoded_data.data() + stream_offsets.offsets[strm_id]
                                            : (col_streams[rg_idx - 1].data_ptrs[strm_type] +
                                               col_streams[rg_idx - 1].lengths[strm_type]);
            } else if (strm_type == gpu::CI_DATA && streams[strm_id].length == 0 &&
                       (ck.type_kind == DOUBLE || ck.type_kind == FLOAT)) {
              // Pass-through
              strm.lengths[strm_type]   = ck.num_rows * ck.dtype_len;
              strm.data_ptrs[strm_type] = nullptr;

            } else if (ck.type_kind == DECIMAL && strm_type == gpu::CI_DATA) {
              strm.lengths[strm_type]   = dec_chunk_sizes.rg_sizes.at(col_idx)[rg_idx];
              strm.data_ptrs[strm_type] = (rg_idx == 0)
                                            ? encoded_data.data() + stream_offsets.offsets[strm_id]
                                            : (col_streams[rg_idx - 1].data_ptrs[strm_type] +
                                               col_streams[rg_idx - 1].lengths[strm_type]);
            } else {
              strm.lengths[strm_type] = RLE_stream_size(streams[strm_id].type_kind, ck.num_rows);
              // RLE encoded streams are stored after all non-RLE streams
              strm.data_ptrs[strm_type] =
                (rg_idx == 0) ? (encoded_data.data() + stream_offsets.non_rle_data_size +
                                 stream_offsets.offsets[strm_id])
                              : (col_streams[rg_idx - 1].data_ptrs[strm_type] +
                                 col_streams[rg_idx - 1].lengths[strm_type]);
            }
          } else {
            strm.lengths[strm_type]   = 0;
            strm.data_ptrs[strm_type] = nullptr;
          }
        }
      }
    }
  }

  chunks.host_to_device(stream);
  chunk_streams.host_to_device(stream);

  gpu::set_chunk_columns(orc_table.d_columns, chunks, stream);

  if (orc_table.num_string_columns() != 0) {
    auto d_stripe_dict = orc_table.string_column(0).device_stripe_dict();
    gpu::EncodeStripeDictionaries(d_stripe_dict,
                                  chunks,
                                  orc_table.num_string_columns(),
                                  segmentation.num_stripes(),
                                  chunk_streams,
                                  stream);
  }

  gpu::EncodeOrcColumnData(chunks, chunk_streams, stream);
  dictionaries.data.clear();
  dictionaries.index.clear();
  stream.synchronize();

  return {std::move(encoded_data), std::move(chunk_streams)};
}

std::vector<StripeInformation> writer::impl::gather_stripes(
  size_t num_index_streams,
  file_segmentation const& segmentation,
  hostdevice_2dvector<gpu::encoder_chunk_streams>* enc_streams,
  hostdevice_2dvector<gpu::StripeStream>* strm_desc)
{
  std::vector<StripeInformation> stripes(segmentation.num_stripes());
  for (auto const& stripe : segmentation.stripes) {
    for (size_t col_idx = 0; col_idx < enc_streams->size().first; col_idx++) {
      const auto& strm = (*enc_streams)[col_idx][stripe.first];

      // Assign stream data of column data stream(s)
      for (int k = 0; k < gpu::CI_INDEX; k++) {
        const auto stream_id = strm.ids[k];
        if (stream_id != -1) {
          auto* ss           = &(*strm_desc)[stripe.id][stream_id - num_index_streams];
          ss->stream_size    = 0;
          ss->first_chunk_id = stripe.first;
          ss->num_chunks     = stripe.size;
          ss->column_id      = col_idx;
          ss->stream_type    = k;
        }
      }
    }

    stripes[stripe.id].numberOfRows =
      stripe.size == 0 ? 0
                       : segmentation.rowgroups[stripe.first + stripe.size - 1][0].end -
                           segmentation.rowgroups[stripe.first][0].begin;
  }

  strm_desc->host_to_device(stream);
  gpu::CompactOrcDataStreams(*strm_desc, *enc_streams, stream);
  strm_desc->device_to_host(stream);
  enc_streams->device_to_host(stream, true);

  return stripes;
}

void set_stat_desc_leaf_cols(device_span<orc_column_device_view const> columns,
                             device_span<stats_column_desc> stat_desc,
                             rmm::cuda_stream_view stream)
{
  thrust::for_each(
    rmm::exec_policy(stream),
    thrust::make_counting_iterator(0ul),
    thrust::make_counting_iterator(stat_desc.size()),
    [=] __device__(auto idx) { stat_desc[idx].leaf_column = &columns[idx].cudf_column; });
}

std::vector<std::vector<uint8_t>> writer::impl::gather_statistic_blobs(
  orc_table_view const& orc_table, file_segmentation const& segmentation)
{
  auto const num_stat_blobs = (1 + segmentation.num_stripes()) * orc_table.num_columns();

  hostdevice_vector<stats_column_desc> stat_desc(orc_table.num_columns(), stream);
  hostdevice_vector<statistics_merge_group> stat_merge(num_stat_blobs, stream);

  for (auto const& column : orc_table.columns) {
    stats_column_desc* desc = &stat_desc[column.index()];
    switch (column.orc_kind()) {
      case TypeKind::BYTE: desc->stats_dtype = dtype_int8; break;
      case TypeKind::SHORT: desc->stats_dtype = dtype_int16; break;
      case TypeKind::INT: desc->stats_dtype = dtype_int32; break;
      case TypeKind::LONG: desc->stats_dtype = dtype_int64; break;
      case TypeKind::FLOAT: desc->stats_dtype = dtype_float32; break;
      case TypeKind::DOUBLE: desc->stats_dtype = dtype_float64; break;
      case TypeKind::BOOLEAN: desc->stats_dtype = dtype_bool; break;
      case TypeKind::DATE: desc->stats_dtype = dtype_int32; break;
      case TypeKind::DECIMAL: desc->stats_dtype = dtype_decimal64; break;
      case TypeKind::TIMESTAMP: desc->stats_dtype = dtype_timestamp64; break;
      case TypeKind::STRING: desc->stats_dtype = dtype_string; break;
      default: desc->stats_dtype = dtype_none; break;
    }
    desc->num_rows   = column.size();
    desc->num_values = column.size();
    if (desc->stats_dtype == dtype_timestamp64) {
      // Timestamp statistics are in milliseconds
      switch (column.scale()) {
        case 9: desc->ts_scale = 1000; break;
        case 6: desc->ts_scale = 0; break;
        case 3: desc->ts_scale = -1000; break;
        case 0: desc->ts_scale = -1000000; break;
        default: desc->ts_scale = 0; break;
      }
    } else {
      desc->ts_scale = 0;
    }
    for (auto const& stripe : segmentation.stripes) {
      auto grp = &stat_merge[column.index() * segmentation.num_stripes() + stripe.id];
      grp->col = stat_desc.device_ptr(column.index());
      grp->start_chunk =
        static_cast<uint32_t>(column.index() * segmentation.num_rowgroups() + stripe.first);
      grp->num_chunks = stripe.size;
    }
    statistics_merge_group* col_stats =
      &stat_merge[segmentation.num_stripes() * orc_table.num_columns() + column.index()];
    col_stats->col         = stat_desc.device_ptr(column.index());
    col_stats->start_chunk = static_cast<uint32_t>(column.index() * segmentation.num_stripes());
    col_stats->num_chunks  = static_cast<uint32_t>(segmentation.num_stripes());
  }
  stat_desc.host_to_device(stream);
  stat_merge.host_to_device(stream);
  set_stat_desc_leaf_cols(orc_table.d_columns, stat_desc, stream);

  auto const num_chunks = segmentation.rowgroups.count();
  rmm::device_uvector<statistics_chunk> stat_chunks(num_chunks + num_stat_blobs, stream);
  rmm::device_uvector<statistics_group> stat_groups(num_chunks, stream);
  gpu::orc_init_statistics_groups(
    stat_groups.data(), stat_desc.device_ptr(), segmentation.rowgroups, stream);

  detail::calculate_group_statistics<detail::io_file_format::ORC>(
    stat_chunks.data(), stat_groups.data(), num_chunks, stream);
  detail::merge_group_statistics<detail::io_file_format::ORC>(
    stat_chunks.data() + num_chunks,
    stat_chunks.data(),
    stat_merge.device_ptr(),
    segmentation.num_stripes() * orc_table.num_columns(),
    stream);

  detail::merge_group_statistics<detail::io_file_format::ORC>(
    stat_chunks.data() + num_chunks + segmentation.num_stripes() * orc_table.num_columns(),
    stat_chunks.data() + num_chunks,
    stat_merge.device_ptr(segmentation.num_stripes() * orc_table.num_columns()),
    orc_table.num_columns(),
    stream);
  gpu::orc_init_statistics_buffersize(
    stat_merge.device_ptr(), stat_chunks.data() + num_chunks, num_stat_blobs, stream);
  stat_merge.device_to_host(stream, true);

  hostdevice_vector<uint8_t> blobs(
    stat_merge[num_stat_blobs - 1].start_chunk + stat_merge[num_stat_blobs - 1].num_chunks, stream);
  gpu::orc_encode_statistics(blobs.device_ptr(),
                             stat_merge.device_ptr(),
                             stat_chunks.data() + num_chunks,
                             num_stat_blobs,
                             stream);
  stat_merge.device_to_host(stream);
  blobs.device_to_host(stream, true);

  std::vector<std::vector<uint8_t>> stat_blobs(num_stat_blobs);
  for (size_t i = 0; i < num_stat_blobs; i++) {
    const uint8_t* stat_begin = blobs.host_ptr(stat_merge[i].start_chunk);
    const uint8_t* stat_end   = stat_begin + stat_merge[i].num_chunks;
    stat_blobs[i].assign(stat_begin, stat_end);
  }

  return stat_blobs;
}

void writer::impl::write_index_stream(int32_t stripe_id,
                                      int32_t stream_id,
                                      host_span<orc_column_view const> columns,
                                      stripe_rowgroups const& rowgroups_range,
                                      host_2dspan<gpu::encoder_chunk_streams const> enc_streams,
                                      host_2dspan<gpu::StripeStream const> strm_desc,
                                      host_span<gpu_inflate_status_s const> comp_out,
                                      StripeInformation* stripe,
                                      orc_streams* streams,
                                      ProtobufWriter* pbw)
{
  row_group_index_info present;
  row_group_index_info data;
  row_group_index_info data2;
  auto kind            = TypeKind::STRUCT;
  auto const column_id = stream_id - 1;

  auto find_record = [=, &strm_desc](gpu::encoder_chunk_streams const& stream,
                                     gpu::StreamIndexType type) {
    row_group_index_info record;
    if (stream.ids[type] > 0) {
      record.pos = 0;
      if (compression_kind_ != NONE) {
        auto const& ss   = strm_desc[stripe_id][stream.ids[type] - (columns.size() + 1)];
        record.blk_pos   = ss.first_block;
        record.comp_pos  = 0;
        record.comp_size = ss.stream_size;
      }
    }
    return record;
  };
  auto scan_record = [=, &comp_out](gpu::encoder_chunk_streams const& stream,
                                    gpu::StreamIndexType type,
                                    row_group_index_info& record) {
    if (record.pos >= 0) {
      record.pos += stream.lengths[type];
      while ((record.pos >= 0) && (record.blk_pos >= 0) &&
             (static_cast<size_t>(record.pos) >= compression_blocksize_) &&
             (record.comp_pos + 3 + comp_out[record.blk_pos].bytes_written <
              static_cast<size_t>(record.comp_size))) {
        record.pos -= compression_blocksize_;
        record.comp_pos += 3 + comp_out[record.blk_pos].bytes_written;
        record.blk_pos += 1;
      }
    }
  };

  // TBD: Not sure we need an empty index stream for column 0
  if (stream_id != 0) {
    const auto& strm = enc_streams[column_id][0];
    present          = find_record(strm, gpu::CI_PRESENT);
    data             = find_record(strm, gpu::CI_DATA);
    data2            = find_record(strm, gpu::CI_DATA2);

    // Change string dictionary to int from index point of view
    kind = columns[column_id].orc_kind();
    if (kind == TypeKind::STRING && columns[column_id].orc_encoding() == DICTIONARY_V2) {
      kind = TypeKind::INT;
    }
  }

  buffer_.resize((compression_kind_ != NONE) ? 3 : 0);

  // Add row index entries
  std::for_each(rowgroups_range.cbegin(), rowgroups_range.cend(), [&](auto rowgroup) {
    pbw->put_row_index_entry(
      present.comp_pos, present.pos, data.comp_pos, data.pos, data2.comp_pos, data2.pos, kind);

    if (stream_id != 0) {
      const auto& strm = enc_streams[column_id][rowgroup];
      scan_record(strm, gpu::CI_PRESENT, present);
      scan_record(strm, gpu::CI_DATA, data);
      scan_record(strm, gpu::CI_DATA2, data2);
    }
  });

  (*streams)[stream_id].length = buffer_.size();
  if (compression_kind_ != NONE) {
    uint32_t uncomp_ix_len = (uint32_t)((*streams)[stream_id].length - 3) * 2 + 1;
    buffer_[0]             = static_cast<uint8_t>(uncomp_ix_len >> 0);
    buffer_[1]             = static_cast<uint8_t>(uncomp_ix_len >> 8);
    buffer_[2]             = static_cast<uint8_t>(uncomp_ix_len >> 16);
  }
  out_sink_->host_write(buffer_.data(), buffer_.size());
  stripe->indexLength += buffer_.size();
}

void writer::impl::write_data_stream(gpu::StripeStream const& strm_desc,
                                     gpu::encoder_chunk_streams const& enc_stream,
                                     uint8_t const* compressed_data,
                                     uint8_t* stream_out,
                                     StripeInformation* stripe,
                                     orc_streams* streams)
{
  const auto length                                        = strm_desc.stream_size;
  (*streams)[enc_stream.ids[strm_desc.stream_type]].length = length;
  if (length == 0) { return; }

  const auto* stream_in = (compression_kind_ == NONE) ? enc_stream.data_ptrs[strm_desc.stream_type]
                                                      : (compressed_data + strm_desc.bfr_offset);

  if (out_sink_->is_device_write_preferred(length)) {
    out_sink_->device_write(stream_in, length, stream);
  } else {
    CUDA_TRY(
      hipMemcpyAsync(stream_out, stream_in, length, hipMemcpyDeviceToHost, stream.value()));
    stream.synchronize();

    out_sink_->host_write(stream_out, length);
  }
  stripe->dataLength += length;
}

void writer::impl::add_uncompressed_block_headers(std::vector<uint8_t>& v)
{
  if (compression_kind_ != NONE) {
    size_t uncomp_len = v.size() - 3, pos = 0, block_len;
    while (uncomp_len > compression_blocksize_) {
      block_len  = compression_blocksize_ * 2 + 1;
      v[pos + 0] = static_cast<uint8_t>(block_len >> 0);
      v[pos + 1] = static_cast<uint8_t>(block_len >> 8);
      v[pos + 2] = static_cast<uint8_t>(block_len >> 16);
      pos += 3 + compression_blocksize_;
      v.insert(v.begin() + pos, 3, 0);
      uncomp_len -= compression_blocksize_;
    }
    block_len  = uncomp_len * 2 + 1;
    v[pos + 0] = static_cast<uint8_t>(block_len >> 0);
    v[pos + 1] = static_cast<uint8_t>(block_len >> 8);
    v[pos + 2] = static_cast<uint8_t>(block_len >> 16);
  }
}

writer::impl::impl(std::unique_ptr<data_sink> sink,
                   orc_writer_options const& options,
                   SingleWriteMode mode,
                   rmm::cuda_stream_view stream,
                   rmm::mr::device_memory_resource* mr)
  : compression_kind_(to_orc_compression(options.get_compression())),
    enable_statistics_(options.enable_statistics()),
    out_sink_(std::move(sink)),
    single_write_mode(mode == SingleWriteMode::YES),
    user_metadata(options.get_metadata()),
    stream(stream),
    _mr(mr)
{
  init_state();
}

writer::impl::impl(std::unique_ptr<data_sink> sink,
                   chunked_orc_writer_options const& options,
                   SingleWriteMode mode,
                   rmm::cuda_stream_view stream,
                   rmm::mr::device_memory_resource* mr)
  : compression_kind_(to_orc_compression(options.get_compression())),
    enable_statistics_(options.enable_statistics()),
    out_sink_(std::move(sink)),
    single_write_mode(mode == SingleWriteMode::YES),
    stream(stream),
    _mr(mr)
{
  if (options.get_metadata() != nullptr) {
    user_metadata_with_nullability = *options.get_metadata();
    user_metadata                  = &user_metadata_with_nullability;
  }

  init_state();
}

writer::impl::~impl() { close(); }

void writer::impl::init_state()
{
  // Write file header
  out_sink_->host_write(MAGIC, std::strlen(MAGIC));
}

/**
 * @brief pre-order append ORC device columns
 */
void __device__ append_orc_device_column(uint32_t& idx,
                                         thrust::optional<uint32_t> parent_idx,
                                         device_span<orc_column_device_view> cols,
                                         column_device_view col)
{
  auto const current_idx = idx;
  cols[current_idx]      = orc_column_device_view{col, parent_idx};
  idx++;
  if (col.type().id() == type_id::LIST) {
    append_orc_device_column(idx, current_idx, cols, col.child(1));
  }
  if (col.type().id() == type_id::STRUCT) {
    for (auto child_idx = 0; child_idx < col.num_child_columns(); ++child_idx) {
      append_orc_device_column(idx, current_idx, cols, col.child(child_idx));
    }
  }
};

orc_table_view make_orc_table_view(table_view const& table,
                                   table_device_view const& d_table,
                                   table_metadata const* user_metadata,
                                   rmm::cuda_stream_view stream)
{
  std::vector<orc_column_view> orc_columns;
  std::vector<uint32_t> str_col_indexes;

  std::function<void(column_view const&, std::optional<int>)> append_orc_column =
    [&](column_view const& col, std::optional<int> index_in_table) {
      auto const str_idx =
        (col.type().id() == type_id::STRING) ? std::optional{str_col_indexes.size()} : std::nullopt;
      auto const& new_col =
        orc_columns.emplace_back(orc_columns.size(), str_idx, index_in_table, col, user_metadata);
      if (new_col.is_string()) { str_col_indexes.push_back(new_col.index()); }
      if (col.type().id() == type_id::LIST) append_orc_column(col.child(1), std::nullopt);
      if (col.type().id() == type_id::STRUCT)
        for (auto child = col.child_begin(); child != col.child_end(); ++child)
          append_orc_column(*child, std::nullopt);
    };

  for (auto col_idx = 0; col_idx < table.num_columns(); ++col_idx) {
    append_orc_column(table.column(col_idx), col_idx);
  }

  rmm::device_uvector<orc_column_device_view> d_orc_columns(orc_columns.size(), stream);

  cudf::detail::device_single_thread(
    [d_orc_cols = device_span<orc_column_device_view>{d_orc_columns},
     d_table    = d_table] __device__() mutable {
      uint32_t idx = 0;
      for (auto const& column : d_table) {
        append_orc_device_column(idx, thrust::nullopt, d_orc_cols, column);
      }
    },
    stream);

  return {std::move(orc_columns),
          std::move(d_orc_columns),
          str_col_indexes,
          cudf::detail::make_device_uvector_sync(str_col_indexes, stream)};
}

hostdevice_2dvector<rowgroup_rows> calculate_rowgroup_bounds(orc_table_view const& orc_table,
                                                             size_type rowgroup_size,
                                                             rmm::cuda_stream_view stream)
{
  auto const num_rowgroups =
    cudf::util::div_rounding_up_unsafe<size_t, size_t>(orc_table.num_rows(), rowgroup_size);

  hostdevice_2dvector<rowgroup_rows> rowgroup_bounds(
    num_rowgroups, orc_table.num_columns(), stream);
  thrust::for_each_n(
    rmm::exec_policy(stream),
    thrust::make_counting_iterator(0ul),
    num_rowgroups,
    [cols      = device_span<orc_column_device_view const>{orc_table.d_columns},
     rg_bounds = device_2dspan<rowgroup_rows>{rowgroup_bounds},
     rowgroup_size] __device__(auto rg_idx) mutable {
      thrust::transform(
        thrust::seq, cols.begin(), cols.end(), rg_bounds[rg_idx].begin(), [&](auto const& col) {
          // Root column
          if (!col.parent_index.has_value()) {
            size_type const rows_begin = rg_idx * rowgroup_size;
            auto const rows_end =
              thrust::min<size_type>((rg_idx + 1) * rowgroup_size, col.cudf_column.size());
            return rowgroup_rows{rows_begin, rows_end};
          } else {
            // Child column
            auto const parent_index       = *col.parent_index;
            column_device_view parent_col = cols[parent_index].cudf_column;
            if (parent_col.type().id() != type_id::LIST) return rg_bounds[rg_idx][parent_index];

            auto parent_offsets = parent_col.child(lists_column_view::offsets_column_index);
            auto const& parent_rowgroup_rows = rg_bounds[rg_idx][parent_index];
            auto const rows_begin = parent_offsets.element<size_type>(parent_rowgroup_rows.begin);
            auto const rows_end   = parent_offsets.element<size_type>(parent_rowgroup_rows.end);
            return rowgroup_rows{rows_begin, rows_end};
          }
        });
    });
  rowgroup_bounds.device_to_host(stream, true);

  return rowgroup_bounds;
}

// returns host vector of per-rowgroup sizes
encoder_decimal_info decimal_chunk_sizes(orc_table_view& orc_table,
                                         file_segmentation const& segmentation,
                                         rmm::cuda_stream_view stream)
{
  std::map<uint32_t, rmm::device_uvector<uint32_t>> elem_sizes;
  // Compute per-element offsets (within each row group) on the device
  for (auto& orc_col : orc_table.columns) {
    if (orc_col.orc_kind() == DECIMAL) {
      auto& current_sizes =
        elem_sizes.insert({orc_col.index(), rmm::device_uvector<uint32_t>(orc_col.size(), stream)})
          .first->second;
      thrust::tabulate(rmm::exec_policy(stream),
                       current_sizes.begin(),
                       current_sizes.end(),
                       [d_cols  = device_span<orc_column_device_view const>{orc_table.d_columns},
                        col_idx = orc_col.index()] __device__(auto idx) {
                         auto const& col = d_cols[col_idx].cudf_column;
                         if (col.is_null(idx)) return 0u;
                         int64_t const element   = (col.type().id() == type_id::DECIMAL32)
                                                     ? col.element<int32_t>(idx)
                                                     : col.element<int64_t>(idx);
                         int64_t const sign      = (element < 0) ? 1 : 0;
                         uint64_t zigzaged_value = ((element ^ -sign) * 2) + sign;

                         uint32_t encoded_length = 1;
                         while (zigzaged_value > 127) {
                           zigzaged_value >>= 7u;
                           ++encoded_length;
                         }
                         return encoded_length;
                       });

      // Compute element offsets within each row group
      thrust::for_each_n(rmm::exec_policy(stream),
                         thrust::make_counting_iterator(0ul),
                         segmentation.num_rowgroups(),
                         [sizes     = device_span<uint32_t>{current_sizes},
                          rg_bounds = device_2dspan<rowgroup_rows const>{segmentation.rowgroups},
                          col_idx   = orc_col.index()] __device__(auto rg_idx) {
                           auto const& range = rg_bounds[rg_idx][col_idx];
                           thrust::inclusive_scan(thrust::seq,
                                                  sizes.begin() + range.begin,
                                                  sizes.begin() + range.end,
                                                  sizes.begin() + range.begin);
                         });

      orc_col.attach_decimal_offsets(current_sizes.data());
    }
  }
  if (elem_sizes.empty()) return {};

  // Gather the row group sizes and copy to host
  auto d_tmp_rowgroup_sizes = rmm::device_uvector<uint32_t>(segmentation.num_rowgroups(), stream);
  std::map<uint32_t, std::vector<uint32_t>> rg_sizes;
  for (auto const& [col_idx, esizes] : elem_sizes) {
    // Copy last elem in each row group - equal to row group size
    thrust::tabulate(rmm::exec_policy(stream),
                     d_tmp_rowgroup_sizes.begin(),
                     d_tmp_rowgroup_sizes.end(),
                     [src       = esizes.data(),
                      col_idx   = col_idx,
                      rg_bounds = device_2dspan<rowgroup_rows const>{
                        segmentation.rowgroups}] __device__(auto idx) {
                       return src[rg_bounds[idx][col_idx].end - 1];
                     });

    rg_sizes[col_idx] = cudf::detail::make_std_vector_async(d_tmp_rowgroup_sizes, stream);
  }

  return {std::move(elem_sizes), std::move(rg_sizes)};
}

std::map<uint32_t, size_t> decimal_column_sizes(
  std::map<uint32_t, std::vector<uint32_t>> const& chunk_sizes)
{
  std::map<uint32_t, size_t> column_sizes;
  std::transform(chunk_sizes.cbegin(),
                 chunk_sizes.cend(),
                 std::inserter(column_sizes, column_sizes.end()),
                 [](auto const& chunk_size) -> std::pair<uint32_t, size_t> {
                   return {
                     chunk_size.first,
                     std::accumulate(chunk_size.second.cbegin(), chunk_size.second.cend(), 0lu)};
                 });
  return column_sizes;
}

string_dictionaries allocate_dictionaries(orc_table_view const& orc_table,
                                          host_2dspan<rowgroup_rows const> rowgroup_bounds,
                                          rmm::cuda_stream_view stream)
{
  thrust::host_vector<bool> is_dict_enabled(orc_table.num_columns());
  for (auto col_idx : orc_table.string_column_indices)
    is_dict_enabled[col_idx] = std::all_of(
      thrust::make_counting_iterator(0ul),
      thrust::make_counting_iterator(rowgroup_bounds.size().first),
      [&](auto rg_idx) {
        return rowgroup_bounds[rg_idx][col_idx].size() < std::numeric_limits<uint16_t>::max();
      });

  std::vector<rmm::device_uvector<uint32_t>> data;
  std::transform(orc_table.string_column_indices.begin(),
                 orc_table.string_column_indices.end(),
                 std::back_inserter(data),
                 [&](auto& idx) {
                   return cudf::detail::make_zeroed_device_uvector_async<uint32_t>(
                     orc_table.columns[idx].size(), stream);
                 });
  std::vector<rmm::device_uvector<uint32_t>> index;
  std::transform(orc_table.string_column_indices.begin(),
                 orc_table.string_column_indices.end(),
                 std::back_inserter(index),
                 [&](auto& idx) {
                   return cudf::detail::make_zeroed_device_uvector_async<uint32_t>(
                     orc_table.columns[idx].size(), stream);
                 });
  stream.synchronize();

  std::vector<device_span<uint32_t>> data_ptrs;
  std::transform(data.begin(), data.end(), std::back_inserter(data_ptrs), [](auto& uvec) {
    return device_span<uint32_t>{uvec};
  });
  std::vector<device_span<uint32_t>> index_ptrs;
  std::transform(index.begin(), index.end(), std::back_inserter(index_ptrs), [](auto& uvec) {
    return device_span<uint32_t>{uvec};
  });

  return {std::move(data),
          std::move(index),
          cudf::detail::make_device_uvector_sync(data_ptrs, stream),
          cudf::detail::make_device_uvector_sync(index_ptrs, stream),
          std::move(is_dict_enabled)};
}

void writer::impl::write(table_view const& table)
{
  CUDF_EXPECTS(not closed, "Data has already been flushed to out and closed");
  auto const num_rows = table.num_rows();

  auto const d_table = table_device_view::create(table, stream);

  auto orc_table = make_orc_table_view(table, *d_table, user_metadata, stream);

  auto rowgroup_bounds = calculate_rowgroup_bounds(orc_table, row_index_stride_, stream);

  // Build per-column dictionary indices
  auto dictionaries          = allocate_dictionaries(orc_table, rowgroup_bounds, stream);
  const auto num_dict_chunks = rowgroup_bounds.size().first * orc_table.num_string_columns();
  hostdevice_vector<gpu::DictionaryChunk> dict(num_dict_chunks, stream);
  if (orc_table.num_string_columns() != 0) {
    init_dictionaries(orc_table,
                      rowgroup_bounds,
                      dictionaries.d_data_view,
                      dictionaries.d_index_view,
                      &dict,
                      stream);
  }

  // Decide stripe boundaries based on rowgroups and dict chunks
  auto const segmentation =
    calculate_segmentation(orc_table.columns, std::move(rowgroup_bounds), max_stripe_size_);

  // Build stripe-level dictionaries
  const auto num_stripe_dict = segmentation.num_stripes() * orc_table.num_string_columns();
  hostdevice_vector<gpu::StripeDictionary> stripe_dict(num_stripe_dict, stream);
  if (orc_table.num_string_columns() != 0) {
    build_dictionaries(orc_table,
                       segmentation.stripes,
                       dict,
                       dictionaries.index,
                       dictionaries.dictionary_enabled,
                       stripe_dict);
  }

  auto dec_chunk_sizes = decimal_chunk_sizes(orc_table, segmentation, stream);

  auto streams =
    create_streams(orc_table.columns, segmentation, decimal_column_sizes(dec_chunk_sizes.rg_sizes));
  auto enc_data = encode_columns(
    orc_table, std::move(dictionaries), std::move(dec_chunk_sizes), segmentation, streams);

  // Assemble individual disparate column chunks into contiguous data streams
  size_type const num_index_streams = (orc_table.num_columns() + 1);
  const auto num_data_streams       = streams.size() - num_index_streams;
  hostdevice_2dvector<gpu::StripeStream> strm_descs(
    segmentation.num_stripes(), num_data_streams, stream);
  auto stripes = gather_stripes(num_index_streams, segmentation, &enc_data.streams, &strm_descs);

  // Gather column statistics
  std::vector<ColStatsBlob> column_stats;
  if (enable_statistics_ && table.num_columns() > 0 && num_rows > 0) {
    column_stats = gather_statistic_blobs(orc_table, segmentation);
  }

  // Allocate intermediate output stream buffer
  size_t compressed_bfr_size   = 0;
  size_t num_compressed_blocks = 0;
  auto stream_output           = [&]() {
    size_t max_stream_size = 0;
    bool all_device_write  = true;

    for (size_t stripe_id = 0; stripe_id < segmentation.num_stripes(); stripe_id++) {
      for (size_t i = 0; i < num_data_streams; i++) {  // TODO range for (at least)
        gpu::StripeStream* ss = &strm_descs[stripe_id][i];
        if (!out_sink_->is_device_write_preferred(ss->stream_size)) { all_device_write = false; }
        size_t stream_size = ss->stream_size;
        if (compression_kind_ != NONE) {
          ss->first_block = num_compressed_blocks;
          ss->bfr_offset  = compressed_bfr_size;

          auto num_blocks = std::max<uint32_t>(
            (stream_size + compression_blocksize_ - 1) / compression_blocksize_, 1);
          stream_size += num_blocks * 3;
          num_compressed_blocks += num_blocks;
          compressed_bfr_size += stream_size;
        }
        max_stream_size = std::max(max_stream_size, stream_size);
      }
    }

    if (all_device_write) {
      return pinned_buffer<uint8_t>{nullptr, hipHostFree};
    } else {
      return pinned_buffer<uint8_t>{[](size_t size) {
                                      uint8_t* ptr = nullptr;
                                      CUDA_TRY(hipHostMalloc(&ptr, size));
                                      return ptr;
                                    }(max_stream_size),
                                    hipHostFree};
    }
  }();

  // Compress the data streams
  rmm::device_buffer compressed_data(compressed_bfr_size, stream);
  hostdevice_vector<gpu_inflate_status_s> comp_out(num_compressed_blocks, stream);
  hostdevice_vector<gpu_inflate_input_s> comp_in(num_compressed_blocks, stream);
  if (compression_kind_ != NONE) {
    strm_descs.host_to_device(stream);
    gpu::CompressOrcDataStreams(static_cast<uint8_t*>(compressed_data.data()),
                                num_compressed_blocks,
                                compression_kind_,
                                compression_blocksize_,
                                strm_descs,
                                enc_data.streams,
                                comp_in.device_ptr(),
                                comp_out.device_ptr(),
                                stream);
    strm_descs.device_to_host(stream);
    comp_out.device_to_host(stream, true);
  }

  ProtobufWriter pbw_(&buffer_);

  // Write stripes
  for (size_t stripe_id = 0; stripe_id < stripes.size(); ++stripe_id) {
    auto const& rowgroups_range = segmentation.stripes[stripe_id];
    auto& stripe                = stripes[stripe_id];

    stripe.offset = out_sink_->bytes_written();

    // Column (skippable) index streams appear at the start of the stripe
    for (size_type stream_id = 0; stream_id < num_index_streams; ++stream_id) {
      write_index_stream(stripe_id,
                         stream_id,
                         orc_table.columns,
                         rowgroups_range,
                         enc_data.streams,
                         strm_descs,
                         comp_out,
                         &stripe,
                         &streams,
                         &pbw_);
    }

    // Column data consisting one or more separate streams
    for (auto const& strm_desc : strm_descs[stripe_id]) {
      write_data_stream(strm_desc,
                        enc_data.streams[strm_desc.column_id][rowgroups_range.first],
                        static_cast<uint8_t*>(compressed_data.data()),
                        stream_output.get(),
                        &stripe,
                        &streams);
    }

    // Write stripefooter consisting of stream information
    StripeFooter sf;
    sf.streams = streams;
    sf.columns.resize(orc_table.num_columns() + 1);
    sf.columns[0].kind = DIRECT;
    for (size_t i = 1; i < sf.columns.size(); ++i) {
      sf.columns[i].kind = orc_table.column(i - 1).orc_encoding();
      sf.columns[i].dictionarySize =
        (sf.columns[i].kind == DICTIONARY_V2)
          ? orc_table.column(i - 1).host_stripe_dict(stripe_id)->num_strings
          : 0;
      if (orc_table.column(i - 1).orc_kind() == TIMESTAMP) { sf.writerTimezone = "UTC"; }
    }
    buffer_.resize((compression_kind_ != NONE) ? 3 : 0);
    pbw_.write(sf);
    stripe.footerLength = buffer_.size();
    if (compression_kind_ != NONE) {
      uint32_t uncomp_sf_len = (stripe.footerLength - 3) * 2 + 1;
      buffer_[0]             = static_cast<uint8_t>(uncomp_sf_len >> 0);
      buffer_[1]             = static_cast<uint8_t>(uncomp_sf_len >> 8);
      buffer_[2]             = static_cast<uint8_t>(uncomp_sf_len >> 16);
    }
    out_sink_->host_write(buffer_.data(), buffer_.size());
  }

  if (column_stats.size() != 0) {
    // File-level statistics
    // NOTE: Excluded from chunked write mode to avoid the need for merging stats across calls
    if (single_write_mode) {
      // First entry contains total number of rows
      buffer_.resize(0);
      pbw_.putb(1 * 8 + PB_TYPE_VARINT);
      pbw_.put_uint(num_rows);
      ff.statistics.reserve(1 + orc_table.num_columns());
      ff.statistics.emplace_back(std::move(buffer_));
      // Add file stats, stored after stripe stats in `column_stats`
      ff.statistics.insert(
        ff.statistics.end(),
        std::make_move_iterator(column_stats.begin()) + stripes.size() * orc_table.num_columns(),
        std::make_move_iterator(column_stats.end()));
    }
    // Stripe-level statistics
    size_t first_stripe = md.stripeStats.size();
    md.stripeStats.resize(first_stripe + stripes.size());
    for (size_t stripe_id = 0; stripe_id < stripes.size(); stripe_id++) {
      md.stripeStats[first_stripe + stripe_id].colStats.resize(1 + orc_table.num_columns());
      buffer_.resize(0);
      pbw_.putb(1 * 8 + PB_TYPE_VARINT);
      pbw_.put_uint(stripes[stripe_id].numberOfRows);
      md.stripeStats[first_stripe + stripe_id].colStats[0] = std::move(buffer_);
      for (size_t col_idx = 0; col_idx < orc_table.num_columns(); col_idx++) {
        size_t idx = stripes.size() * col_idx + stripe_id;
        if (idx < column_stats.size()) {
          md.stripeStats[first_stripe + stripe_id].colStats[1 + col_idx] =
            std::move(column_stats[idx]);
        }
      }
    }
  }
  if (ff.headerLength == 0) {
    // First call
    ff.headerLength   = std::strlen(MAGIC);
    ff.rowIndexStride = row_index_stride_;
    ff.types.resize(1 + orc_table.num_columns());
    ff.types[0].kind = STRUCT;
    for (auto const& column : orc_table.columns) {
      if (!column.is_child()) {
        ff.types[0].subtypes.emplace_back(column.id());
        ff.types[0].fieldNames.emplace_back(column.orc_name());
      }
    }
    for (auto const& column : orc_table.columns) {
      auto& schema_type = ff.types[column.id()];
      schema_type.kind  = column.orc_kind();
      if (column.orc_kind() == DECIMAL) {
        schema_type.scale     = static_cast<uint32_t>(column.scale());
        schema_type.precision = column.precision();
      }
      // In preorder tracersal the column after a list column is always the child column
      if (column.orc_kind() == LIST) { schema_type.subtypes.emplace_back(column.id() + 1); }
    }
  } else {
    // verify the user isn't passing mismatched tables
    CUDF_EXPECTS(ff.types.size() == 1 + orc_table.num_columns(),
                 "Mismatch in table structure between multiple calls to write");
    CUDF_EXPECTS(
      std::all_of(orc_table.columns.cbegin(),
                  orc_table.columns.cend(),
                  [&](auto const& col) { return ff.types[col.id()].kind == col.orc_kind(); }),
      "Mismatch in column types between multiple calls to write");
  }
  ff.stripes.insert(ff.stripes.end(),
                    std::make_move_iterator(stripes.begin()),
                    std::make_move_iterator(stripes.end()));
  ff.numberOfRows += num_rows;
}

void writer::impl::close()
{
  if (closed) { return; }
  closed = true;
  ProtobufWriter pbw_(&buffer_);
  PostScript ps;

  ff.contentLength = out_sink_->bytes_written();
  if (user_metadata) {
    for (auto it = user_metadata->user_data.begin(); it != user_metadata->user_data.end(); it++) {
      ff.metadata.push_back({it->first, it->second});
    }
  }
  // Write statistics metadata
  if (md.stripeStats.size() != 0) {
    buffer_.resize((compression_kind_ != NONE) ? 3 : 0);
    pbw_.write(md);
    add_uncompressed_block_headers(buffer_);
    ps.metadataLength = buffer_.size();
    out_sink_->host_write(buffer_.data(), buffer_.size());
  } else {
    ps.metadataLength = 0;
  }
  buffer_.resize((compression_kind_ != NONE) ? 3 : 0);
  pbw_.write(ff);
  add_uncompressed_block_headers(buffer_);

  // Write postscript metadata
  ps.footerLength         = buffer_.size();
  ps.compression          = compression_kind_;
  ps.compressionBlockSize = compression_blocksize_;
  ps.version              = {0, 12};
  ps.magic                = MAGIC;
  const auto ps_length    = static_cast<uint8_t>(pbw_.write(ps));
  buffer_.push_back(ps_length);
  out_sink_->host_write(buffer_.data(), buffer_.size());
  out_sink_->flush();
}

// Forward to implementation
writer::writer(std::unique_ptr<data_sink> sink,
               orc_writer_options const& options,
               SingleWriteMode mode,
               rmm::cuda_stream_view stream,
               rmm::mr::device_memory_resource* mr)
  : _impl(std::make_unique<impl>(std::move(sink), options, mode, stream, mr))
{
}

// Forward to implementation
writer::writer(std::unique_ptr<data_sink> sink,
               chunked_orc_writer_options const& options,
               SingleWriteMode mode,
               rmm::cuda_stream_view stream,
               rmm::mr::device_memory_resource* mr)
  : _impl(std::make_unique<impl>(std::move(sink), options, mode, stream, mr))
{
}

// Destructor within this translation unit
writer::~writer() = default;

// Forward to implementation
void writer::write(table_view const& table) { _impl->write(table); }

// Forward to implementation
void writer::close() { _impl->close(); }

}  // namespace orc
}  // namespace detail
}  // namespace io
}  // namespace cudf
