#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019-2021, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

/**
 * @file writer_impl.cu
 * @brief cuDF-IO ORC writer class implementation
 */

#include "writer_impl.hpp"

#include <io/utilities/column_utils.cuh>

#include <cudf/null_mask.hpp>
#include <cudf/strings/strings_column_view.hpp>
#include <cudf/utilities/span.hpp>

#include <rmm/cuda_stream_view.hpp>
#include <rmm/device_buffer.hpp>
#include <rmm/device_uvector.hpp>

#include <algorithm>
#include <cstring>
#include <numeric>
#include <utility>

namespace cudf {
namespace io {
namespace detail {
namespace orc {
using namespace cudf::io::orc;
using namespace cudf::io;

struct row_group_index_info {
  int32_t pos       = -1;  // Position
  int32_t blk_pos   = -1;  // Block Position
  int32_t comp_pos  = -1;  // Compressed Position
  int32_t comp_size = -1;  // Compressed size
};

namespace {
/**
 * @brief Helper for pinned host memory
 */
template <typename T>
using pinned_buffer = std::unique_ptr<T, decltype(&hipHostFree)>;

/**
 * @brief Function that translates GDF compression to ORC compression
 */
orc::CompressionKind to_orc_compression(compression_type compression)
{
  switch (compression) {
    case compression_type::AUTO:
    case compression_type::SNAPPY: return orc::CompressionKind::SNAPPY;
    case compression_type::NONE: return orc::CompressionKind::NONE;
    default: CUDF_EXPECTS(false, "Unsupported compression type"); return orc::CompressionKind::NONE;
  }
}

/**
 * @brief Function that translates GDF dtype to ORC datatype
 */
constexpr orc::TypeKind to_orc_type(cudf::type_id id)
{
  switch (id) {
    case cudf::type_id::INT8: return TypeKind::BYTE;
    case cudf::type_id::INT16: return TypeKind::SHORT;
    case cudf::type_id::INT32: return TypeKind::INT;
    case cudf::type_id::INT64: return TypeKind::LONG;
    case cudf::type_id::FLOAT32: return TypeKind::FLOAT;
    case cudf::type_id::FLOAT64: return TypeKind::DOUBLE;
    case cudf::type_id::BOOL8: return TypeKind::BOOLEAN;
    case cudf::type_id::TIMESTAMP_DAYS: return TypeKind::DATE;
    case cudf::type_id::TIMESTAMP_SECONDS:
    case cudf::type_id::TIMESTAMP_MICROSECONDS:
    case cudf::type_id::TIMESTAMP_MILLISECONDS:
    case cudf::type_id::TIMESTAMP_NANOSECONDS: return TypeKind::TIMESTAMP;
    case cudf::type_id::STRING: return TypeKind::STRING;
    default: return TypeKind::INVALID_TYPE_KIND;
  }
}

/**
 * @brief Function that translates time unit to nanoscale multiple
 */
template <typename T>
constexpr T to_clockscale(cudf::type_id timestamp_id)
{
  switch (timestamp_id) {
    case cudf::type_id::TIMESTAMP_SECONDS: return 9;
    case cudf::type_id::TIMESTAMP_MILLISECONDS: return 6;
    case cudf::type_id::TIMESTAMP_MICROSECONDS: return 3;
    case cudf::type_id::TIMESTAMP_NANOSECONDS:
    default: return 0;
  }
}

}  // namespace

/**
 * @brief Helper class that adds ORC-specific column info
 */
class orc_column_view {
 public:
  /**
   * @brief Constructor that extracts out the string position + length pairs
   * for building dictionaries for string columns
   */
  explicit orc_column_view(size_t id,
                           size_t str_id,
                           column_view const &col,
                           const table_metadata *metadata,
                           rmm::cuda_stream_view stream)
    : _id(id),
      _str_id(str_id),
      _is_string_type(col.type().id() == type_id::STRING),
      _type_width(_is_string_type ? 0 : cudf::size_of(col.type())),
      _data_count(col.size()),
      _null_count(col.null_count()),
      _nulls(col.null_mask()),
      _clockscale(to_clockscale<uint8_t>(col.type().id())),
      _type_kind(to_orc_type(col.type().id()))
  {
    // Generating default name if name isn't present in metadata
    if (metadata && _id < metadata->column_names.size()) {
      _name = metadata->column_names[_id];
    } else {
      _name = "_col" + std::to_string(_id);
    }
  }

  auto is_string() const noexcept { return _is_string_type; }
  void set_dict_stride(size_t stride) noexcept { dict_stride = stride; }
  auto get_dict_stride() const noexcept { return dict_stride; }

  /**
   * @brief Function that associates an existing dictionary chunk allocation
   */
  void attach_dict_chunk(gpu::DictionaryChunk *host_dict, gpu::DictionaryChunk *dev_dict)
  {
    dict   = host_dict;
    d_dict = dev_dict;
  }
  auto host_dict_chunk(size_t rowgroup) const
  {
    assert(_is_string_type);
    return &dict[rowgroup * dict_stride + _str_id];
  }
  auto device_dict_chunk() const { return d_dict; }

  /**
   * @brief Function that associates an existing stripe dictionary allocation
   */
  void attach_stripe_dict(gpu::StripeDictionary *host_stripe_dict,
                          gpu::StripeDictionary *dev_stripe_dict)
  {
    stripe_dict   = host_stripe_dict;
    d_stripe_dict = dev_stripe_dict;
  }
  auto host_stripe_dict(size_t stripe) const
  {
    assert(_is_string_type);
    return &stripe_dict[stripe * dict_stride + _str_id];
  }
  auto device_stripe_dict() const { return d_stripe_dict; }

  auto id() const noexcept { return _id; }
  size_t type_width() const noexcept { return _type_width; }
  size_t data_count() const noexcept { return _data_count; }
  size_t null_count() const noexcept { return _null_count; }
  bool nullable() const noexcept { return (_nulls != nullptr); }
  uint32_t const *nulls() const noexcept { return _nulls; }
  uint8_t clockscale() const noexcept { return _clockscale; }

  void set_orc_encoding(ColumnEncodingKind e) { _encoding_kind = e; }
  auto orc_kind() const noexcept { return _type_kind; }
  auto orc_encoding() const noexcept { return _encoding_kind; }
  auto orc_name() const noexcept { return _name; }

 private:
  // Identifier within set of columns and string columns, respectively
  size_t _id           = 0;
  size_t _str_id       = 0;
  bool _is_string_type = false;

  size_t _type_width     = 0;
  size_t _data_count     = 0;
  size_t _null_count     = 0;
  uint32_t const *_nulls = nullptr;
  uint8_t _clockscale    = 0;

  // ORC-related members
  std::string _name{};
  TypeKind _type_kind;
  ColumnEncodingKind _encoding_kind;

  // String dictionary-related members
  size_t dict_stride                       = 0;
  gpu::DictionaryChunk const *dict         = nullptr;
  gpu::StripeDictionary const *stripe_dict = nullptr;
  gpu::DictionaryChunk *d_dict             = nullptr;
  gpu::StripeDictionary *d_stripe_dict     = nullptr;
};

std::vector<stripe_rowgroups> writer::impl::gather_stripe_info(
  host_span<orc_column_view const> columns, size_t num_rowgroups)
{
  auto const is_any_column_string =
    std::any_of(columns.begin(), columns.end(), [](auto const &col) { return col.is_string(); });
  // Apply rows per stripe limit to limit string dictionaries
  size_t const max_stripe_rows = is_any_column_string ? 1000000 : 5000000;

  std::vector<stripe_rowgroups> infos;
  for (size_t rowgroup = 0, stripe_start = 0, stripe_size = 0; rowgroup < num_rowgroups;
       ++rowgroup) {
    auto const rowgroup_size =
      std::accumulate(columns.begin(), columns.end(), 0ul, [&](size_t total_size, auto const &col) {
        if (col.is_string()) {
          const auto dt = col.host_dict_chunk(rowgroup);
          return total_size + row_index_stride_ + dt->string_char_count;
        } else {
          return total_size + col.type_width() * row_index_stride_;
        }
      });

    if ((rowgroup > stripe_start) &&
        (stripe_size + rowgroup_size > max_stripe_size_ ||
         (rowgroup + 1 - stripe_start) * row_index_stride_ > max_stripe_rows)) {
      infos.emplace_back(infos.size(), stripe_start, rowgroup - stripe_start);
      stripe_start = rowgroup;
      stripe_size  = 0;
    }
    stripe_size += rowgroup_size;
    if (rowgroup + 1 == num_rowgroups) {
      infos.emplace_back(infos.size(), stripe_start, num_rowgroups - stripe_start);
    }
  }

  return infos;
}

void writer::impl::init_dictionaries(const table_device_view &view,
                                     orc_column_view *columns,
                                     std::vector<int> const &str_col_ids,
                                     device_span<size_type> d_str_col_ids,
                                     uint32_t *dict_data,
                                     uint32_t *dict_index,
                                     hostdevice_vector<gpu::DictionaryChunk> *dict)
{
  const size_t num_rowgroups = dict->size() / str_col_ids.size();

  // Setup per-rowgroup dictionary indexes for each dictionary-aware column
  for (size_t i = 0; i < str_col_ids.size(); ++i) {
    auto &str_column = columns[str_col_ids[i]];
    str_column.set_dict_stride(str_col_ids.size());
    str_column.attach_dict_chunk(dict->host_ptr(), dict->device_ptr());
  }

  gpu::InitDictionaryIndices(view,
                             dict->device_ptr(),
                             dict_data,
                             dict_index,
                             row_index_stride_,
                             d_str_col_ids.data(),
                             d_str_col_ids.size(),
                             num_rowgroups,
                             stream);
  dict->device_to_host(stream, true);
}

void writer::impl::build_dictionaries(orc_column_view *columns,
                                      std::vector<int> const &str_col_ids,
                                      host_span<stripe_rowgroups const> stripe_bounds,
                                      hostdevice_vector<gpu::DictionaryChunk> const &dict,
                                      uint32_t *dict_index,
                                      hostdevice_vector<gpu::StripeDictionary> &stripe_dict)
{
  const auto num_rowgroups = dict.size() / str_col_ids.size();

  for (size_t col_idx = 0; col_idx < str_col_ids.size(); ++col_idx) {
    auto &str_column = columns[str_col_ids[col_idx]];
    str_column.attach_stripe_dict(stripe_dict.host_ptr(), stripe_dict.device_ptr());

    for (auto const &stripe : stripe_bounds) {
      auto &sd           = stripe_dict[stripe.id * str_col_ids.size() + col_idx];
      sd.dict_data       = str_column.host_dict_chunk(stripe.first)->dict_data;
      sd.dict_index      = dict_index + col_idx * str_column.data_count();  // Indexed by abs row
      sd.column_id       = str_col_ids[col_idx];
      sd.start_chunk     = stripe.first;
      sd.num_chunks      = stripe.size;
      sd.dict_char_count = 0;
      sd.num_strings =
        std::accumulate(stripe.cbegin(), stripe.cend(), 0, [&](auto dt_str_cnt, auto rg_idx) {
          const auto &dt = dict[rg_idx * str_col_ids.size() + col_idx];
          return dt_str_cnt + dt.num_dict_strings;
        });
      sd.leaf_column = dict[col_idx].leaf_column;
    }

    if (enable_dictionary_) {
      struct string_column_cost {
        size_t direct     = 0;
        size_t dictionary = 0;
      };
      auto const col_cost =
        std::accumulate(stripe_bounds.front().cbegin(),
                        stripe_bounds.back().cend(),
                        string_column_cost{},
                        [&](auto cost, auto rg_idx) -> string_column_cost {
                          const auto &dt = dict[rg_idx * str_col_ids.size() + col_idx];
                          return {cost.direct + dt.string_char_count,
                                  cost.dictionary + dt.dict_char_count + dt.num_dict_strings};
                        });
      // Disable dictionary if it does not reduce the output size
      if (col_cost.dictionary >= col_cost.direct) {
        for (auto const &stripe : stripe_bounds) {
          stripe_dict[stripe.id * str_col_ids.size() + col_idx].dict_data = nullptr;
        }
      }
    }
  }

  stripe_dict.host_to_device(stream);
  gpu::BuildStripeDictionaries(stripe_dict.device_ptr(),
                               stripe_dict.host_ptr(),
                               dict.device_ptr(),
                               stripe_bounds.size(),
                               num_rowgroups,
                               str_col_ids.size(),
                               stream);
  stripe_dict.device_to_host(stream, true);
}

orc_streams writer::impl::create_streams(host_span<orc_column_view> columns,
                                         host_span<stripe_rowgroups const> stripe_bounds)
{
  // First n + 1 streams are row index streams, including 'column 0'
  std::vector<Stream> streams{{ROW_INDEX, 0, 0}};  // TODO: Separate index and data streams?
  streams.resize(columns.size() + 1);
  std::vector<int32_t> ids(columns.size() * gpu::CI_NUM_STREAMS, -1);

  for (auto &column : columns) {
    TypeKind kind                    = column.orc_kind();
    StreamKind data_kind             = DATA;
    StreamKind data2_kind            = LENGTH;
    ColumnEncodingKind encoding_kind = DIRECT;

    int64_t present_stream_size = 0;
    int64_t data_stream_size    = 0;
    int64_t data2_stream_size   = 0;
    int64_t dict_stream_size    = 0;

    auto const is_nullable = [&]() {
      if (single_write_mode) {
        return column.nullable();
      } else {
        return (column.id() < user_metadata_with_nullability.column_nullable.size())
                 ? user_metadata_with_nullability.column_nullable[column.id()]
                 : true;
      }
    }();
    if (is_nullable) {
      present_stream_size = ((row_index_stride_ + 7) >> 3);
      present_stream_size += (present_stream_size + 0x7f) >> 7;
    }

    switch (kind) {
      case TypeKind::BOOLEAN:
        data_stream_size = div_rowgroups_by<int64_t>(1024) * (128 + 1);
        encoding_kind    = DIRECT;
        break;
      case TypeKind::BYTE:
        data_stream_size = div_rowgroups_by<int64_t>(128) * (128 + 1);
        encoding_kind    = DIRECT;
        break;
      case TypeKind::SHORT:
        data_stream_size = div_rowgroups_by<int64_t>(512) * (512 * 2 + 2);
        encoding_kind    = DIRECT_V2;
        break;
      case TypeKind::FLOAT:
        // Pass through if no nulls (no RLE encoding for floating point)
        data_stream_size =
          (column.null_count() != 0) ? div_rowgroups_by<int64_t>(512) * (512 * 4 + 2) : INT64_C(-1);
        encoding_kind = DIRECT;
        break;
      case TypeKind::INT:
      case TypeKind::DATE:
        data_stream_size = div_rowgroups_by<int64_t>(512) * (512 * 4 + 2);
        encoding_kind    = DIRECT_V2;
        break;
      case TypeKind::DOUBLE:
        // Pass through if no nulls (no RLE encoding for floating point)
        data_stream_size =
          (column.null_count() != 0) ? div_rowgroups_by<int64_t>(512) * (512 * 8 + 2) : INT64_C(-1);
        encoding_kind = DIRECT;
        break;
      case TypeKind::LONG:
        data_stream_size = div_rowgroups_by<int64_t>(512) * (512 * 8 + 2);
        encoding_kind    = DIRECT_V2;
        break;
      case TypeKind::STRING: {
        bool enable_dict           = enable_dictionary_;
        size_t dict_data_size      = 0;
        size_t dict_strings        = 0;
        size_t dict_lengths_div512 = 0;
        for (auto const &stripe : stripe_bounds) {
          const auto sd = column.host_stripe_dict(stripe.id);
          enable_dict   = (enable_dict && sd->dict_data != nullptr);
          if (enable_dict) {
            dict_strings += sd->num_strings;
            dict_lengths_div512 += (sd->num_strings + 0x1ff) >> 9;
            dict_data_size += sd->dict_char_count;
          }
        }

        auto const direct_data_size =
          std::accumulate(stripe_bounds.front().cbegin(),
                          stripe_bounds.back().cend(),
                          size_t{0},
                          [&](auto data_size, auto rg_idx) {
                            return data_size + column.host_dict_chunk(rg_idx)->string_char_count;
                          });
        if (enable_dict) {
          uint32_t dict_bits = 0;
          for (dict_bits = 1; dict_bits < 32; dict_bits <<= 1) {
            if (dict_strings <= (1ull << dict_bits)) break;
          }
          const auto valid_count = column.data_count() - column.null_count();
          dict_data_size += (dict_bits * valid_count + 7) >> 3;
        }

        // Decide between direct or dictionary encoding
        if (enable_dict && dict_data_size < direct_data_size) {
          data_stream_size  = div_rowgroups_by<int64_t>(512) * (512 * 4 + 2);
          data2_stream_size = dict_lengths_div512 * (512 * 4 + 2);
          dict_stream_size  = std::max<size_t>(dict_data_size, 1);
          encoding_kind     = DICTIONARY_V2;
        } else {
          data_stream_size  = std::max<size_t>(direct_data_size, 1);
          data2_stream_size = div_rowgroups_by<int64_t>(512) * (512 * 4 + 2);
          encoding_kind     = DIRECT_V2;
        }
        break;
      }
      case TypeKind::TIMESTAMP:
        data_stream_size  = ((row_index_stride_ + 0x1ff) >> 9) * (512 * 4 + 2);
        data2_stream_size = data_stream_size;
        data2_kind        = SECONDARY;
        encoding_kind     = DIRECT_V2;
        break;
      default: CUDF_FAIL("Unsupported ORC type kind");
    }

    // Initialize the column's metadata (this is the only reason columns is in/out param)
    column.set_orc_encoding(encoding_kind);

    // Initialize the column's index stream
    const auto id      = static_cast<uint32_t>(1 + column.id());
    streams[id].column = id;
    streams[id].kind   = ROW_INDEX;
    streams[id].length = 0;

    // Initialize the column's data stream(s)
    const auto base = column.id() * gpu::CI_NUM_STREAMS;
    if (present_stream_size != 0) {
      auto len                    = static_cast<uint64_t>(present_stream_size);
      ids[base + gpu::CI_PRESENT] = streams.size();
      streams.push_back(orc::Stream{PRESENT, id, len});
    }
    if (data_stream_size != 0) {
      auto len                 = static_cast<uint64_t>(std::max<int64_t>(data_stream_size, 0));
      ids[base + gpu::CI_DATA] = streams.size();
      streams.push_back(orc::Stream{data_kind, id, len});
    }
    if (data2_stream_size != 0) {
      auto len                  = static_cast<uint64_t>(std::max<int64_t>(data2_stream_size, 0));
      ids[base + gpu::CI_DATA2] = streams.size();
      streams.push_back(orc::Stream{data2_kind, id, len});
    }
    if (dict_stream_size != 0) {
      auto len                       = static_cast<uint64_t>(dict_stream_size);
      ids[base + gpu::CI_DICTIONARY] = streams.size();
      streams.push_back(orc::Stream{DICTIONARY_DATA, id, len});
    }
  }
  return {std::move(streams), std::move(ids)};
}

orc_streams::orc_stream_offsets orc_streams::compute_offsets(
  host_span<orc_column_view const> columns, size_t num_rowgroups) const
{
  std::vector<size_t> strm_offsets(streams.size());
  size_t str_data_size = 0;
  size_t rle_data_size = 0;
  for (size_t i = 0; i < streams.size(); ++i) {
    const auto &stream = streams[i];
    const auto &column = columns[stream.column - 1];

    if (((stream.kind == DICTIONARY_DATA || stream.kind == LENGTH) &&
         (column.orc_encoding() == DICTIONARY_V2)) ||
        ((stream.kind == DATA) &&
         (column.orc_kind() == TypeKind::STRING && column.orc_encoding() == DIRECT_V2))) {
      strm_offsets[i] = str_data_size;
      str_data_size += stream.length;
    } else {
      strm_offsets[i] = rle_data_size;
      rle_data_size += (stream.length * num_rowgroups + 7) & ~7;
    }
  }
  str_data_size = (str_data_size + 7) & ~7;

  return {std::move(strm_offsets), str_data_size, rle_data_size};
}

struct segmented_valid_cnt_input {
  bitmask_type const *mask;
  std::vector<size_type> indices;
};

encoded_data writer::impl::encode_columns(const table_device_view &view,
                                          host_span<orc_column_view const> columns,
                                          std::vector<int> const &str_col_ids,
                                          host_span<stripe_rowgroups const> stripe_bounds,
                                          orc_streams const &streams)
{
  auto const num_columns   = columns.size();
  auto const num_rowgroups = stripes_size(stripe_bounds);
  hostdevice_2dvector<gpu::EncChunk> chunks(num_columns, num_rowgroups, stream);
  hostdevice_2dvector<gpu::encoder_chunk_streams> chunk_streams(num_columns, num_rowgroups, stream);
  auto const stream_offsets = streams.compute_offsets(columns, num_rowgroups);
  rmm::device_uvector<uint8_t> encoded_data(stream_offsets.data_size(), stream);

  // Initialize column chunks' descriptions
  std::map<size_type, segmented_valid_cnt_input> validity_check_inputs;

  for (auto const &column : columns) {
    for (auto const &stripe : stripe_bounds) {
      for (auto rg_idx_it = stripe.cbegin(); rg_idx_it < stripe.cend(); ++rg_idx_it) {
        auto const rg_idx = *rg_idx_it;
        auto &ck          = chunks[column.id()][rg_idx];

        ck.start_row = (rg_idx * row_index_stride_);
        ck.num_rows  = std::min<uint32_t>(row_index_stride_, column.data_count() - ck.start_row);
        ck.encoding_kind = column.orc_encoding();
        ck.type_kind     = column.orc_kind();
        if (ck.type_kind == TypeKind::STRING) {
          ck.dict_index = (ck.encoding_kind == DICTIONARY_V2)
                            ? column.host_stripe_dict(stripe.id)->dict_index
                            : nullptr;
          ck.dtype_len = 1;
        } else {
          ck.dtype_len = column.type_width();
        }
        ck.scale = column.clockscale();
        // Only need to check row groups that end within the stripe
      }
    }
  }

  auto validity_check_indices = [&](size_t col_idx) {
    std::vector<size_type> indices;
    for (auto const &stripe : stripe_bounds) {
      for (auto rg_idx_it = stripe.cbegin(); rg_idx_it < stripe.cend() - 1; ++rg_idx_it) {
        auto const &chunk = chunks[col_idx][*rg_idx_it];
        indices.push_back(chunk.start_row);
        indices.push_back(chunk.start_row + chunk.num_rows);
      }
    }
    return indices;
  };
  for (auto const &column : columns) {
    if (column.orc_kind() == TypeKind::BOOLEAN && column.nullable()) {
      validity_check_inputs[column.id()] = {column.nulls(), validity_check_indices(column.id())};
    }
  }
  for (auto &cnt_in : validity_check_inputs) {
    auto const valid_counts = segmented_count_set_bits(cnt_in.second.mask, cnt_in.second.indices);
    CUDF_EXPECTS(std::none_of(valid_counts.cbegin(),
                              valid_counts.cend(),
                              [](auto valid_count) { return valid_count % 8; }),
                 "There's currently a bug in encoding boolean columns. Suggested workaround "
                 "is to convert "
                 "to "
                 "int8 type. Please see https://github.com/rapidsai/cudf/issues/6763 for "
                 "more information.");
  }

  for (size_t col_idx = 0; col_idx < num_columns; col_idx++) {
    auto const &column = columns[col_idx];
    auto col_streams   = chunk_streams[col_idx];
    for (auto const &stripe : stripe_bounds) {
      for (auto rg_idx_it = stripe.cbegin(); rg_idx_it < stripe.cend(); ++rg_idx_it) {
        auto const rg_idx = *rg_idx_it;
        auto const &ck    = chunks[col_idx][rg_idx];
        auto &strm        = col_streams[rg_idx];

        for (int strm_type = 0; strm_type < gpu::CI_NUM_STREAMS; ++strm_type) {
          auto const strm_id = streams.id(col_idx * gpu::CI_NUM_STREAMS + strm_type);

          strm.ids[strm_type] = strm_id;
          if (strm_id >= 0) {
            if ((strm_type == gpu::CI_DICTIONARY) ||
                (strm_type == gpu::CI_DATA2 && ck.encoding_kind == DICTIONARY_V2)) {
              if (rg_idx_it == stripe.cbegin()) {
                const int32_t dict_stride = column.get_dict_stride();
                const auto stripe_dict    = column.host_stripe_dict(stripe.id);
                strm.lengths[strm_type] =
                  (strm_type == gpu::CI_DICTIONARY)
                    ? stripe_dict->dict_char_count
                    : (((stripe_dict->num_strings + 0x1ff) >> 9) * (512 * 4 + 2));
                if (stripe.id == 0) {
                  strm.data_ptrs[strm_type] = encoded_data.data() + stream_offsets.offsets[strm_id];
                } else {
                  auto const &strm_up = col_streams[stripe_dict[-dict_stride].start_chunk];
                  strm.data_ptrs[strm_type] =
                    strm_up.data_ptrs[strm_type] + strm_up.lengths[strm_type];
                }
              } else {
                strm.lengths[strm_type]   = 0;
                strm.data_ptrs[strm_type] = col_streams[rg_idx - 1].data_ptrs[strm_type];
              }
            } else if (strm_type == gpu::CI_DATA && ck.type_kind == TypeKind::STRING &&
                       ck.encoding_kind == DIRECT_V2) {
              strm.lengths[strm_type] = column.host_dict_chunk(rg_idx)->string_char_count;
              auto const &prev_strm   = col_streams[rg_idx - 1];
              strm.data_ptrs[strm_type] =
                (rg_idx == 0) ? encoded_data.data() + stream_offsets.offsets[strm_id]
                              : (prev_strm.data_ptrs[strm_type] + prev_strm.lengths[strm_type]);
            } else if (strm_type == gpu::CI_DATA && streams[strm_id].length == 0 &&
                       (ck.type_kind == DOUBLE || ck.type_kind == FLOAT)) {
              // Pass-through
              strm.lengths[strm_type]   = ck.num_rows * ck.dtype_len;
              strm.data_ptrs[strm_type] = nullptr;
            } else {
              strm.lengths[strm_type]   = streams[strm_id].length;
              strm.data_ptrs[strm_type] = encoded_data.data() + stream_offsets.str_data_size +
                                          stream_offsets.offsets[strm_id] +
                                          streams[strm_id].length * rg_idx;
            }
          } else {
            strm.lengths[strm_type]   = 0;
            strm.data_ptrs[strm_type] = nullptr;
          }
        }
      }
    }
  }

  chunks.host_to_device(stream);
  chunk_streams.host_to_device(stream);

  gpu::set_chunk_columns(view, chunks, stream);

  if (!str_col_ids.empty()) {
    auto d_stripe_dict = columns[str_col_ids[0]].device_stripe_dict();
    gpu::EncodeStripeDictionaries(
      d_stripe_dict, chunks, str_col_ids.size(), stripe_bounds.size(), chunk_streams, stream);
  }

  gpu::EncodeOrcColumnData(chunks, chunk_streams, stream);
  stream.synchronize();

  return {std::move(encoded_data), std::move(chunk_streams)};
}

std::vector<StripeInformation> writer::impl::gather_stripes(
  size_t num_rows,
  size_t num_index_streams,
  host_span<stripe_rowgroups const> stripe_bounds,
  hostdevice_2dvector<gpu::encoder_chunk_streams> *enc_streams,
  hostdevice_2dvector<gpu::StripeStream> *strm_desc)
{
  std::vector<StripeInformation> stripes(stripe_bounds.size());
  for (auto const &stripe : stripe_bounds) {
    for (size_t col_idx = 0; col_idx < enc_streams->size().first; col_idx++) {
      const auto &strm = (*enc_streams)[col_idx][stripe.first];

      // Assign stream data of column data stream(s)
      for (int k = 0; k < gpu::CI_INDEX; k++) {
        const auto stream_id = strm.ids[k];
        if (stream_id != -1) {
          auto *ss           = &(*strm_desc)[stripe.id][stream_id - num_index_streams];
          ss->stream_size    = 0;
          ss->first_chunk_id = stripe.first;
          ss->num_chunks     = stripe.size;
          ss->column_id      = col_idx;
          ss->stream_type    = k;
        }
      }
    }

    auto const stripe_group_end     = *stripe.cend();
    auto const stripe_end           = std::min(stripe_group_end * row_index_stride_, num_rows);
    stripes[stripe.id].numberOfRows = stripe_end - stripe.first * row_index_stride_;
  }

  strm_desc->host_to_device(stream);
  gpu::CompactOrcDataStreams(*strm_desc, *enc_streams, stream);
  strm_desc->device_to_host(stream);
  enc_streams->device_to_host(stream, true);

  return stripes;
}

std::vector<std::vector<uint8_t>> writer::impl::gather_statistic_blobs(
  const table_device_view &table,
  host_span<orc_column_view const> columns,
  host_span<stripe_rowgroups const> stripe_bounds)
{
  auto const num_rowgroups = stripes_size(stripe_bounds);
  size_t num_stat_blobs    = (1 + stripe_bounds.size()) * columns.size();
  size_t num_chunks        = num_rowgroups * columns.size();

  std::vector<std::vector<uint8_t>> stat_blobs(num_stat_blobs);
  hostdevice_vector<stats_column_desc> stat_desc(columns.size(), stream);
  hostdevice_vector<statistics_merge_group> stat_merge(num_stat_blobs, stream);
  rmm::device_uvector<statistics_chunk> stat_chunks(num_chunks + num_stat_blobs, stream);
  rmm::device_uvector<statistics_group> stat_groups(num_chunks, stream);

  for (auto const &column : columns) {
    stats_column_desc *desc = &stat_desc[column.id()];
    switch (column.orc_kind()) {
      case TypeKind::BYTE: desc->stats_dtype = dtype_int8; break;
      case TypeKind::SHORT: desc->stats_dtype = dtype_int16; break;
      case TypeKind::INT: desc->stats_dtype = dtype_int32; break;
      case TypeKind::LONG: desc->stats_dtype = dtype_int64; break;
      case TypeKind::FLOAT: desc->stats_dtype = dtype_float32; break;
      case TypeKind::DOUBLE: desc->stats_dtype = dtype_float64; break;
      case TypeKind::BOOLEAN: desc->stats_dtype = dtype_bool; break;
      case TypeKind::DATE: desc->stats_dtype = dtype_int32; break;
      case TypeKind::TIMESTAMP: desc->stats_dtype = dtype_timestamp64; break;
      case TypeKind::STRING: desc->stats_dtype = dtype_string; break;
      default: desc->stats_dtype = dtype_none; break;
    }
    desc->num_rows   = column.data_count();
    desc->num_values = column.data_count();
    if (desc->stats_dtype == dtype_timestamp64) {
      // Timestamp statistics are in milliseconds
      switch (column.clockscale()) {
        case 9: desc->ts_scale = 1000; break;
        case 6: desc->ts_scale = 0; break;
        case 3: desc->ts_scale = -1000; break;
        case 0: desc->ts_scale = -1000000; break;
        default: desc->ts_scale = 0; break;
      }
    } else {
      desc->ts_scale = 0;
    }
    for (auto const &stripe : stripe_bounds) {
      auto grp         = &stat_merge[column.id() * stripe_bounds.size() + stripe.id];
      grp->col         = stat_desc.device_ptr(column.id());
      grp->start_chunk = static_cast<uint32_t>(column.id() * num_rowgroups + stripe.first);
      grp->num_chunks  = stripe.size;
    }
    statistics_merge_group *col_stats =
      &stat_merge[stripe_bounds.size() * columns.size() + column.id()];
    col_stats->col         = stat_desc.device_ptr(column.id());
    col_stats->start_chunk = static_cast<uint32_t>(column.id() * stripe_bounds.size());
    col_stats->num_chunks  = static_cast<uint32_t>(stripe_bounds.size());
  }
  stat_desc.host_to_device(stream);
  stat_merge.host_to_device(stream);

  rmm::device_uvector<column_device_view> leaf_column_views =
    create_leaf_column_device_views<stats_column_desc>(stat_desc, table, stream);

  gpu::orc_init_statistics_groups(stat_groups.data(),
                                  stat_desc.device_ptr(),
                                  columns.size(),
                                  num_rowgroups,
                                  row_index_stride_,
                                  stream);

  GatherColumnStatistics(stat_chunks.data(), stat_groups.data(), num_chunks, stream);
  MergeColumnStatistics(stat_chunks.data() + num_chunks,
                        stat_chunks.data(),
                        stat_merge.device_ptr(),
                        stripe_bounds.size() * columns.size(),
                        stream);

  MergeColumnStatistics(stat_chunks.data() + num_chunks + stripe_bounds.size() * columns.size(),
                        stat_chunks.data() + num_chunks,
                        stat_merge.device_ptr(stripe_bounds.size() * columns.size()),
                        columns.size(),
                        stream);
  gpu::orc_init_statistics_buffersize(
    stat_merge.device_ptr(), stat_chunks.data() + num_chunks, num_stat_blobs, stream);
  stat_merge.device_to_host(stream, true);

  hostdevice_vector<uint8_t> blobs(
    stat_merge[num_stat_blobs - 1].start_chunk + stat_merge[num_stat_blobs - 1].num_chunks, stream);
  gpu::orc_encode_statistics(blobs.device_ptr(),
                             stat_merge.device_ptr(),
                             stat_chunks.data() + num_chunks,
                             num_stat_blobs,
                             stream);
  stat_merge.device_to_host(stream);
  blobs.device_to_host(stream, true);

  for (size_t i = 0; i < num_stat_blobs; i++) {
    const uint8_t *stat_begin = blobs.host_ptr(stat_merge[i].start_chunk);
    const uint8_t *stat_end   = stat_begin + stat_merge[i].num_chunks;
    stat_blobs[i].assign(stat_begin, stat_end);
  }

  return stat_blobs;
}

void writer::impl::write_index_stream(int32_t stripe_id,
                                      int32_t stream_id,
                                      host_span<orc_column_view const> columns,
                                      stripe_rowgroups const &rowgroups_range,
                                      host_2dspan<gpu::encoder_chunk_streams const> enc_streams,
                                      host_2dspan<gpu::StripeStream const> strm_desc,
                                      host_span<gpu_inflate_status_s const> comp_out,
                                      StripeInformation *stripe,
                                      orc_streams *streams,
                                      ProtobufWriter *pbw)
{
  row_group_index_info present;
  row_group_index_info data;
  row_group_index_info data2;
  auto kind            = TypeKind::STRUCT;
  auto const column_id = stream_id - 1;

  auto find_record = [=, &strm_desc](gpu::encoder_chunk_streams const &stream,
                                     gpu::StreamIndexType type) {
    row_group_index_info record;
    if (stream.ids[type] > 0) {
      record.pos = 0;
      if (compression_kind_ != NONE) {
        auto const &ss   = strm_desc[stripe_id][stream.ids[type] - (columns.size() + 1)];
        record.blk_pos   = ss.first_block;
        record.comp_pos  = 0;
        record.comp_size = ss.stream_size;
      }
    }
    return record;
  };
  auto scan_record = [=, &comp_out](gpu::encoder_chunk_streams const &stream,
                                    gpu::StreamIndexType type,
                                    row_group_index_info &record) {
    if (record.pos >= 0) {
      record.pos += stream.lengths[type];
      while ((record.pos >= 0) && (record.blk_pos >= 0) &&
             (static_cast<size_t>(record.pos) >= compression_blocksize_) &&
             (record.comp_pos + 3 + comp_out[record.blk_pos].bytes_written <
              static_cast<size_t>(record.comp_size))) {
        record.pos -= compression_blocksize_;
        record.comp_pos += 3 + comp_out[record.blk_pos].bytes_written;
        record.blk_pos += 1;
      }
    }
  };

  // TBD: Not sure we need an empty index stream for column 0
  if (stream_id != 0) {
    const auto &strm = enc_streams[column_id][0];
    present          = find_record(strm, gpu::CI_PRESENT);
    data             = find_record(strm, gpu::CI_DATA);
    data2            = find_record(strm, gpu::CI_DATA2);

    // Change string dictionary to int from index point of view
    kind = columns[column_id].orc_kind();
    if (kind == TypeKind::STRING && columns[column_id].orc_encoding() == DICTIONARY_V2) {
      kind = TypeKind::INT;
    }
  }

  buffer_.resize((compression_kind_ != NONE) ? 3 : 0);

  // Add row index entries
  std::for_each(rowgroups_range.cbegin(), rowgroups_range.cend(), [&](auto rowgroup) {
    pbw->put_row_index_entry(
      present.comp_pos, present.pos, data.comp_pos, data.pos, data2.comp_pos, data2.pos, kind);

    if (stream_id != 0) {
      const auto &strm = enc_streams[column_id][rowgroup];
      scan_record(strm, gpu::CI_PRESENT, present);
      scan_record(strm, gpu::CI_DATA, data);
      scan_record(strm, gpu::CI_DATA2, data2);
    }
  });

  (*streams)[stream_id].length = buffer_.size();
  if (compression_kind_ != NONE) {
    uint32_t uncomp_ix_len = (uint32_t)((*streams)[stream_id].length - 3) * 2 + 1;
    buffer_[0]             = static_cast<uint8_t>(uncomp_ix_len >> 0);
    buffer_[1]             = static_cast<uint8_t>(uncomp_ix_len >> 8);
    buffer_[2]             = static_cast<uint8_t>(uncomp_ix_len >> 16);
  }
  out_sink_->host_write(buffer_.data(), buffer_.size());
  stripe->indexLength += buffer_.size();
}

void writer::impl::write_data_stream(gpu::StripeStream const &strm_desc,
                                     gpu::encoder_chunk_streams const &enc_stream,
                                     uint8_t const *compressed_data,
                                     uint8_t *stream_out,
                                     StripeInformation *stripe,
                                     orc_streams *streams)
{
  const auto length                                        = strm_desc.stream_size;
  (*streams)[enc_stream.ids[strm_desc.stream_type]].length = length;
  if (length != 0) {
    const auto *stream_in = (compression_kind_ == NONE)
                              ? enc_stream.data_ptrs[strm_desc.stream_type]
                              : (compressed_data + strm_desc.bfr_offset);
    CUDA_TRY(
      hipMemcpyAsync(stream_out, stream_in, length, hipMemcpyDeviceToHost, stream.value()));
    stream.synchronize();

    out_sink_->host_write(stream_out, length);
  }
  stripe->dataLength += length;
}

void writer::impl::add_uncompressed_block_headers(std::vector<uint8_t> &v)
{
  if (compression_kind_ != NONE) {
    size_t uncomp_len = v.size() - 3, pos = 0, block_len;
    while (uncomp_len > compression_blocksize_) {
      block_len  = compression_blocksize_ * 2 + 1;
      v[pos + 0] = static_cast<uint8_t>(block_len >> 0);
      v[pos + 1] = static_cast<uint8_t>(block_len >> 8);
      v[pos + 2] = static_cast<uint8_t>(block_len >> 16);
      pos += 3 + compression_blocksize_;
      v.insert(v.begin() + pos, 3, 0);
      uncomp_len -= compression_blocksize_;
    }
    block_len  = uncomp_len * 2 + 1;
    v[pos + 0] = static_cast<uint8_t>(block_len >> 0);
    v[pos + 1] = static_cast<uint8_t>(block_len >> 8);
    v[pos + 2] = static_cast<uint8_t>(block_len >> 16);
  }
}

writer::impl::impl(std::unique_ptr<data_sink> sink,
                   orc_writer_options const &options,
                   SingleWriteMode mode,
                   rmm::mr::device_memory_resource *mr,
                   rmm::cuda_stream_view stream)
  : compression_kind_(to_orc_compression(options.get_compression())),
    enable_statistics_(options.enable_statistics()),
    out_sink_(std::move(sink)),
    single_write_mode(mode == SingleWriteMode::YES),
    user_metadata(options.get_metadata()),
    stream(stream),
    _mr(mr)
{
  init_state();
}

writer::impl::impl(std::unique_ptr<data_sink> sink,
                   chunked_orc_writer_options const &options,
                   SingleWriteMode mode,
                   rmm::mr::device_memory_resource *mr,
                   rmm::cuda_stream_view stream)
  : compression_kind_(to_orc_compression(options.get_compression())),
    enable_statistics_(options.enable_statistics()),
    out_sink_(std::move(sink)),
    single_write_mode(mode == SingleWriteMode::YES),
    stream(stream),
    _mr(mr)
{
  if (options.get_metadata() != nullptr) {
    user_metadata_with_nullability = *options.get_metadata();
    user_metadata                  = &user_metadata_with_nullability;
  }

  init_state();
}

writer::impl::~impl() { close(); }

void writer::impl::init_state()
{
  // Write file header
  out_sink_->host_write(MAGIC, std::strlen(MAGIC));
}

rmm::device_uvector<size_type> get_string_column_ids(const table_device_view &view,
                                                     rmm::cuda_stream_view stream)
{
  rmm::device_uvector<size_type> string_column_ids(view.num_columns(), stream);
  auto iter     = thrust::make_counting_iterator<size_type>(0);
  auto end_iter = thrust::copy_if(rmm::exec_policy(stream),
                                  iter,
                                  iter + view.num_columns(),
                                  string_column_ids.begin(),
                                  [view] __device__(size_type index) {
                                    return (view.column(index).type().id() == type_id::STRING);
                                  });
  string_column_ids.resize(end_iter - string_column_ids.begin(), stream);
  return string_column_ids;
}

void writer::impl::write(table_view const &table)
{
  CUDF_EXPECTS(not closed, "Data has already been flushed to out and closed");
  auto const num_columns = table.num_columns();
  auto const num_rows    = table.num_rows();

  if (user_metadata_with_nullability.column_nullable.size() > 0) {
    CUDF_EXPECTS(
      user_metadata_with_nullability.column_nullable.size() == static_cast<size_t>(num_columns),
      "When passing values in user_metadata_with_nullability, data for all columns must "
      "be specified");
  }

  auto device_columns    = table_device_view::create(table, stream);
  auto string_column_ids = get_string_column_ids(*device_columns, stream);

  // Wrapper around cudf columns to attach ORC-specific type info
  std::vector<orc_column_view> orc_columns;
  orc_columns.reserve(num_columns);
  // Mapping of string columns for quick look-up
  std::vector<int> str_col_ids;
  for (auto const &column : table) {
    auto const current_id     = orc_columns.size();
    auto const current_str_id = str_col_ids.size();

    orc_columns.emplace_back(current_id, current_str_id, column, user_metadata, stream);
    if (orc_columns.back().is_string()) { str_col_ids.push_back(current_id); }
  }

  rmm::device_uvector<uint32_t> dict_index(str_col_ids.size() * num_rows, stream);
  rmm::device_uvector<uint32_t> dict_data(str_col_ids.size() * num_rows, stream);

  // Build per-column dictionary indices
  const auto num_rowgroups   = div_by_rowgroups<size_t>(num_rows);
  const auto num_dict_chunks = num_rowgroups * str_col_ids.size();
  hostdevice_vector<gpu::DictionaryChunk> dict(num_dict_chunks, stream);
  if (!str_col_ids.empty()) {
    init_dictionaries(*device_columns,
                      orc_columns.data(),
                      str_col_ids,
                      string_column_ids,
                      dict_data.data(),
                      dict_index.data(),
                      &dict);
  }

  // Decide stripe boundaries early on, based on uncompressed size
  auto const stripe_bounds = gather_stripe_info(orc_columns, num_rowgroups);

  // Build stripe-level dictionaries
  const auto num_stripe_dict = stripe_bounds.size() * str_col_ids.size();
  hostdevice_vector<gpu::StripeDictionary> stripe_dict(num_stripe_dict, stream);
  if (!str_col_ids.empty()) {
    build_dictionaries(
      orc_columns.data(), str_col_ids, stripe_bounds, dict, dict_index.data(), stripe_dict);
  }

  auto streams  = create_streams(orc_columns, stripe_bounds);
  auto enc_data = encode_columns(*device_columns, orc_columns, str_col_ids, stripe_bounds, streams);

  // Assemble individual disparate column chunks into contiguous data streams
  const auto num_index_streams = (num_columns + 1);
  const auto num_data_streams  = streams.size() - num_index_streams;
  hostdevice_2dvector<gpu::StripeStream> strm_descs(stripe_bounds.size(), num_data_streams, stream);
  auto stripes =
    gather_stripes(num_rows, num_index_streams, stripe_bounds, &enc_data.streams, &strm_descs);

  // Gather column statistics
  std::vector<std::vector<uint8_t>> column_stats;
  if (enable_statistics_ && num_columns > 0 && num_rows > 0) {
    column_stats = gather_statistic_blobs(*device_columns, orc_columns, stripe_bounds);
  }

  // Allocate intermediate output stream buffer
  size_t compressed_bfr_size   = 0;
  size_t num_compressed_blocks = 0;
  auto stream_output           = [&]() {
    size_t max_stream_size = 0;

    for (size_t stripe_id = 0; stripe_id < stripe_bounds.size(); stripe_id++) {
      for (size_t i = 0; i < num_data_streams; i++) {  // TODO range for (at least)
        gpu::StripeStream *ss = &strm_descs[stripe_id][i];
        size_t stream_size    = ss->stream_size;
        if (compression_kind_ != NONE) {
          ss->first_block = num_compressed_blocks;
          ss->bfr_offset  = compressed_bfr_size;

          auto num_blocks = std::max<uint32_t>(
            (stream_size + compression_blocksize_ - 1) / compression_blocksize_, 1);
          stream_size += num_blocks * 3;
          num_compressed_blocks += num_blocks;
          compressed_bfr_size += stream_size;
        }
        max_stream_size = std::max(max_stream_size, stream_size);
      }
    }

    return pinned_buffer<uint8_t>{[](size_t size) {
                                    uint8_t *ptr = nullptr;
                                    CUDA_TRY(hipHostMalloc(&ptr, size));
                                    return ptr;
                                  }(max_stream_size),
                                  hipHostFree};
  }();

  // Compress the data streams
  rmm::device_buffer compressed_data(compressed_bfr_size, stream);
  hostdevice_vector<gpu_inflate_status_s> comp_out(num_compressed_blocks, stream);
  hostdevice_vector<gpu_inflate_input_s> comp_in(num_compressed_blocks, stream);
  if (compression_kind_ != NONE) {
    strm_descs.host_to_device(stream);
    gpu::CompressOrcDataStreams(static_cast<uint8_t *>(compressed_data.data()),
                                num_compressed_blocks,
                                compression_kind_,
                                compression_blocksize_,
                                strm_descs,
                                enc_data.streams,
                                comp_in.device_ptr(),
                                comp_out.device_ptr(),
                                stream);
    strm_descs.device_to_host(stream);
    comp_out.device_to_host(stream, true);
  }

  ProtobufWriter pbw_(&buffer_);

  // Write stripes
  for (size_t stripe_id = 0; stripe_id < stripes.size(); ++stripe_id) {
    auto const &rowgroup_range = stripe_bounds[stripe_id];
    auto &stripe               = stripes[stripe_id];

    stripe.offset = out_sink_->bytes_written();

    // Column (skippable) index streams appear at the start of the stripe
    for (size_type stream_id = 0; stream_id <= num_columns; ++stream_id) {
      write_index_stream(stripe_id,
                         stream_id,
                         orc_columns,
                         rowgroup_range,
                         enc_data.streams,
                         strm_descs,
                         comp_out,
                         &stripe,
                         &streams,
                         &pbw_);
    }

    // Column data consisting one or more separate streams
    for (auto const &strm_desc : strm_descs[stripe_id]) {
      write_data_stream(strm_desc,
                        enc_data.streams[strm_desc.column_id][rowgroup_range.first],
                        static_cast<uint8_t *>(compressed_data.data()),
                        stream_output.get(),
                        &stripe,
                        &streams);
    }

    // Write stripefooter consisting of stream information
    StripeFooter sf;
    sf.streams = streams;
    sf.columns.resize(num_columns + 1);
    sf.columns[0].kind           = DIRECT;
    sf.columns[0].dictionarySize = 0;
    for (size_t i = 1; i < sf.columns.size(); ++i) {
      sf.columns[i].kind           = orc_columns[i - 1].orc_encoding();
      sf.columns[i].dictionarySize = (sf.columns[i].kind == DICTIONARY_V2)
                                       ? orc_columns[i - 1].host_stripe_dict(stripe_id)->num_strings
                                       : 0;
      if (orc_columns[i - 1].orc_kind() == TIMESTAMP) { sf.writerTimezone = "UTC"; }
    }
    buffer_.resize((compression_kind_ != NONE) ? 3 : 0);
    pbw_.write(sf);
    stripe.footerLength = buffer_.size();
    if (compression_kind_ != NONE) {
      uint32_t uncomp_sf_len = (stripe.footerLength - 3) * 2 + 1;
      buffer_[0]             = static_cast<uint8_t>(uncomp_sf_len >> 0);
      buffer_[1]             = static_cast<uint8_t>(uncomp_sf_len >> 8);
      buffer_[2]             = static_cast<uint8_t>(uncomp_sf_len >> 16);
    }
    out_sink_->host_write(buffer_.data(), buffer_.size());
  }

  if (column_stats.size() != 0) {
    // File-level statistics
    // NOTE: Excluded from chunked write mode to avoid the need for merging stats across calls
    if (single_write_mode) {
      ff.statistics.resize(1 + num_columns);
      // First entry contains total number of rows
      buffer_.resize(0);
      pbw_.putb(1 * 8 + PB_TYPE_VARINT);
      pbw_.put_uint(num_rows);
      ff.statistics[0] = std::move(buffer_);
      for (int col_idx = 0; col_idx < num_columns; col_idx++) {
        size_t idx = stripes.size() * num_columns + col_idx;
        if (idx < column_stats.size()) {
          ff.statistics[1 + col_idx] = std::move(column_stats[idx]);
        }
      }
    }
    // Stripe-level statistics
    size_t first_stripe = md.stripeStats.size();
    md.stripeStats.resize(first_stripe + stripes.size());
    for (size_t stripe_id = 0; stripe_id < stripes.size(); stripe_id++) {
      md.stripeStats[first_stripe + stripe_id].colStats.resize(1 + num_columns);
      buffer_.resize(0);
      pbw_.putb(1 * 8 + PB_TYPE_VARINT);
      pbw_.put_uint(stripes[stripe_id].numberOfRows);
      md.stripeStats[first_stripe + stripe_id].colStats[0] = std::move(buffer_);
      for (int col_idx = 0; col_idx < num_columns; col_idx++) {
        size_t idx = stripes.size() * col_idx + stripe_id;
        if (idx < column_stats.size()) {
          md.stripeStats[first_stripe + stripe_id].colStats[1 + col_idx] =
            std::move(column_stats[idx]);
        }
      }
    }
  }
  if (ff.headerLength == 0) {
    // First call
    ff.headerLength   = std::strlen(MAGIC);
    ff.rowIndexStride = row_index_stride_;
    ff.types.resize(1 + num_columns);
    ff.types[0].kind = STRUCT;
    ff.types[0].subtypes.resize(num_columns);
    ff.types[0].fieldNames.resize(num_columns);
    for (auto const &column : orc_columns) {
      ff.types[1 + column.id()].kind      = column.orc_kind();
      ff.types[0].subtypes[column.id()]   = 1 + column.id();
      ff.types[0].fieldNames[column.id()] = column.orc_name();
    }
  } else {
    // verify the user isn't passing mismatched tables
    CUDF_EXPECTS(ff.types.size() == 1 + orc_columns.size(),
                 "Mismatch in table structure between multiple calls to write");
    CUDF_EXPECTS(
      std::all_of(orc_columns.cbegin(),
                  orc_columns.cend(),
                  [&](auto const &col) { return ff.types[1 + col.id()].kind == col.orc_kind(); }),
      "Mismatch in column types between multiple calls to write");
  }
  ff.stripes.insert(ff.stripes.end(),
                    std::make_move_iterator(stripes.begin()),
                    std::make_move_iterator(stripes.end()));
  ff.numberOfRows += num_rows;
}

void writer::impl::close()
{
  if (closed) { return; }
  closed = true;
  ProtobufWriter pbw_(&buffer_);
  PostScript ps;

  ff.contentLength = out_sink_->bytes_written();
  if (user_metadata) {
    for (auto it = user_metadata->user_data.begin(); it != user_metadata->user_data.end(); it++) {
      ff.metadata.push_back({it->first, it->second});
    }
  }
  // Write statistics metadata
  if (md.stripeStats.size() != 0) {
    buffer_.resize((compression_kind_ != NONE) ? 3 : 0);
    pbw_.write(md);
    add_uncompressed_block_headers(buffer_);
    ps.metadataLength = buffer_.size();
    out_sink_->host_write(buffer_.data(), buffer_.size());
  } else {
    ps.metadataLength = 0;
  }
  buffer_.resize((compression_kind_ != NONE) ? 3 : 0);
  pbw_.write(ff);
  add_uncompressed_block_headers(buffer_);

  // Write postscript metadata
  ps.footerLength         = buffer_.size();
  ps.compression          = compression_kind_;
  ps.compressionBlockSize = compression_blocksize_;
  ps.version              = {0, 12};
  ps.magic                = MAGIC;
  const auto ps_length    = static_cast<uint8_t>(pbw_.write(ps));
  buffer_.push_back(ps_length);
  out_sink_->host_write(buffer_.data(), buffer_.size());
  out_sink_->flush();
}

// Forward to implementation
writer::writer(std::unique_ptr<data_sink> sink,
               orc_writer_options const &options,
               SingleWriteMode mode,
               rmm::mr::device_memory_resource *mr,
               rmm::cuda_stream_view stream)
  : _impl(std::make_unique<impl>(std::move(sink), options, mode, mr, stream))
{
}

// Forward to implementation
writer::writer(std::unique_ptr<data_sink> sink,
               chunked_orc_writer_options const &options,
               SingleWriteMode mode,
               rmm::mr::device_memory_resource *mr,
               rmm::cuda_stream_view stream)
  : _impl(std::make_unique<impl>(std::move(sink), options, mode, mr, stream))
{
}

// Destructor within this translation unit
writer::~writer() = default;

// Forward to implementation
void writer::write(table_view const &table) { _impl->write(table); }

// Forward to implementation
void writer::close() { _impl->close(); }

}  // namespace orc
}  // namespace detail
}  // namespace io
}  // namespace cudf
