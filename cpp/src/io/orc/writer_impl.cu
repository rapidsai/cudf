#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019-2022, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

/**
 * @file writer_impl.cu
 * @brief cuDF-IO ORC writer class implementation
 */

#include "writer_impl.hpp"

#include <io/comp/nvcomp_adapter.hpp>
#include <io/statistics/column_statistics.cuh>
#include <io/utilities/column_utils.cuh>

#include <cudf/detail/iterator.cuh>
#include <cudf/detail/null_mask.hpp>
#include <cudf/detail/utilities/cuda.cuh>
#include <cudf/detail/utilities/vector_factories.hpp>
#include <cudf/strings/strings_column_view.hpp>
#include <cudf/utilities/bit.hpp>
#include <cudf/utilities/span.hpp>

#include <rmm/cuda_stream_view.hpp>
#include <rmm/device_buffer.hpp>
#include <rmm/device_uvector.hpp>

#include <thrust/execution_policy.h>
#include <thrust/extrema.h>
#include <thrust/for_each.h>
#include <thrust/functional.h>
#include <thrust/host_vector.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/iterator/reverse_iterator.h>
#include <thrust/iterator/transform_iterator.h>
#include <thrust/optional.h>
#include <thrust/pair.h>
#include <thrust/reduce.h>
#include <thrust/scan.h>
#include <thrust/tabulate.h>
#include <thrust/transform.h>

#include <algorithm>
#include <cstring>
#include <numeric>
#include <utility>

#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/memcpy_async.h>

#include <cuda/std/climits>
#include <cuda/std/limits>

namespace cudf {
namespace io {
namespace detail {
namespace orc {
using namespace cudf::io::orc;
using namespace cudf::io;

struct row_group_index_info {
  int32_t pos       = -1;  // Position
  int32_t blk_pos   = -1;  // Block Position
  int32_t comp_pos  = -1;  // Compressed Position
  int32_t comp_size = -1;  // Compressed size
};

namespace {
/**
 * @brief Helper for pinned host memory
 */
template <typename T>
using pinned_buffer = std::unique_ptr<T, decltype(&hipHostFree)>;

/**
 * @brief Translates ORC compression to nvCOMP compression
 */
auto to_nvcomp_compression_type(CompressionKind compression_kind)
{
  if (compression_kind == SNAPPY) return nvcomp::compression_type::SNAPPY;
  if (compression_kind == ZLIB) return nvcomp::compression_type::DEFLATE;
  if (compression_kind == ZSTD) return nvcomp::compression_type::ZSTD;
  CUDF_FAIL("Unsupported compression type");
}

/**
 * @brief Translates cuDF compression to ORC compression
 */
orc::CompressionKind to_orc_compression(compression_type compression)
{
  switch (compression) {
    case compression_type::AUTO:
    case compression_type::SNAPPY: return orc::CompressionKind::SNAPPY;
    case compression_type::ZLIB: return orc::CompressionKind::ZLIB;
    case compression_type::ZSTD: return orc::CompressionKind::ZSTD;
    case compression_type::NONE: return orc::CompressionKind::NONE;
    default: CUDF_FAIL("Unsupported compression type");
  }
}

/**
 * @brief Returns the block size for a given compression kind.
 */
constexpr size_t compression_block_size(orc::CompressionKind compression)
{
  if (compression == orc::CompressionKind::NONE) { return 0; }

  auto const ncomp_type   = to_nvcomp_compression_type(compression);
  auto const nvcomp_limit = nvcomp::is_compression_disabled(ncomp_type)
                              ? std::nullopt
                              : nvcomp::compress_max_allowed_chunk_size(ncomp_type);

  constexpr size_t max_block_size = 256 * 1024;
  return std::min(nvcomp_limit.value_or(max_block_size), max_block_size);
}

/**
 * @brief Translates cuDF dtype to ORC datatype
 */
constexpr orc::TypeKind to_orc_type(cudf::type_id id, bool list_column_as_map)
{
  switch (id) {
    case cudf::type_id::INT8: return TypeKind::BYTE;
    case cudf::type_id::INT16: return TypeKind::SHORT;
    case cudf::type_id::INT32: return TypeKind::INT;
    case cudf::type_id::INT64: return TypeKind::LONG;
    case cudf::type_id::FLOAT32: return TypeKind::FLOAT;
    case cudf::type_id::FLOAT64: return TypeKind::DOUBLE;
    case cudf::type_id::BOOL8: return TypeKind::BOOLEAN;
    case cudf::type_id::TIMESTAMP_DAYS: return TypeKind::DATE;
    case cudf::type_id::TIMESTAMP_SECONDS:
    case cudf::type_id::TIMESTAMP_MICROSECONDS:
    case cudf::type_id::TIMESTAMP_MILLISECONDS:
    case cudf::type_id::TIMESTAMP_NANOSECONDS: return TypeKind::TIMESTAMP;
    case cudf::type_id::STRING: return TypeKind::STRING;
    case cudf::type_id::DECIMAL32:
    case cudf::type_id::DECIMAL64:
    case cudf::type_id::DECIMAL128: return TypeKind::DECIMAL;
    case cudf::type_id::LIST: return list_column_as_map ? TypeKind::MAP : TypeKind::LIST;
    case cudf::type_id::STRUCT: return TypeKind::STRUCT;
    default: return TypeKind::INVALID_TYPE_KIND;
  }
}

/**
 * @brief Translates time unit to nanoscale multiple.
 */
constexpr int32_t to_clockscale(cudf::type_id timestamp_id)
{
  switch (timestamp_id) {
    case cudf::type_id::TIMESTAMP_SECONDS: return 9;
    case cudf::type_id::TIMESTAMP_MILLISECONDS: return 6;
    case cudf::type_id::TIMESTAMP_MICROSECONDS: return 3;
    case cudf::type_id::TIMESTAMP_NANOSECONDS:
    default: return 0;
  }
}

/**
 * @brief Returns the precision of the given decimal type.
 */
constexpr auto orc_precision(cudf::type_id decimal_id)
{
  using namespace numeric;
  switch (decimal_id) {
    case cudf::type_id::DECIMAL32: return cuda::std::numeric_limits<decimal32::rep>::digits10;
    case cudf::type_id::DECIMAL64: return cuda::std::numeric_limits<decimal64::rep>::digits10;
    case cudf::type_id::DECIMAL128: return cuda::std::numeric_limits<decimal128::rep>::digits10;
    default: return 0;
  }
}

}  // namespace

/**
 * @brief Helper class that adds ORC-specific column info
 */
class orc_column_view {
 public:
  /**
   * @brief Constructor that extracts out the string position + length pairs
   * for building dictionaries for string columns
   */
  explicit orc_column_view(uint32_t index,
                           int str_idx,
                           orc_column_view* parent,
                           column_view const& col,
                           column_in_metadata const& metadata)
    : cudf_column{col},
      _index{index},
      _str_idx{str_idx},
      _is_child{parent != nullptr},
      _type_width{cudf::is_fixed_width(col.type()) ? cudf::size_of(col.type()) : 0},
      _type_kind{to_orc_type(col.type().id(), metadata.is_map())},
      _scale{(_type_kind == TypeKind::DECIMAL) ? -col.type().scale()
                                               : to_clockscale(col.type().id())},
      _precision{metadata.is_decimal_precision_set() ? metadata.get_decimal_precision()
                                                     : orc_precision(col.type().id())},
      name{metadata.get_name()}
  {
    if (metadata.is_nullability_defined()) { nullable_from_metadata = metadata.nullable(); }
    if (parent != nullptr) {
      parent->add_child(_index);
      _parent_index = parent->index();
    }

    if (_type_kind == TypeKind::MAP) {
      auto const struct_col = col.child(lists_column_view::child_column_index);
      CUDF_EXPECTS(struct_col.null_count() == 0,
                   "struct column of a MAP column should not have null elements");
      CUDF_EXPECTS(struct_col.num_children() == 2, "MAP column must have two child columns");
    }
  }

  void add_child(uint32_t child_idx) { children.emplace_back(child_idx); }

  auto type() const noexcept { return cudf_column.type(); }
  auto is_string() const noexcept { return cudf_column.type().id() == type_id::STRING; }
  void set_dict_stride(size_t stride) noexcept { _dict_stride = stride; }
  [[nodiscard]] auto dict_stride() const noexcept { return _dict_stride; }

  /**
   * @brief Function that associates an existing dictionary chunk allocation
   */
  void attach_dict_chunk(gpu::DictionaryChunk const* host_dict,
                         gpu::DictionaryChunk const* dev_dict)
  {
    dict   = host_dict;
    d_dict = dev_dict;
  }
  [[nodiscard]] auto host_dict_chunk(size_t rowgroup) const
  {
    CUDF_EXPECTS(is_string(), "Dictionary chunks are only present in string columns.");
    return &dict[rowgroup * _dict_stride + _str_idx];
  }
  [[nodiscard]] auto device_dict_chunk() const { return d_dict; }

  [[nodiscard]] auto const& decimal_offsets() const { return d_decimal_offsets; }
  void attach_decimal_offsets(uint32_t* sizes_ptr) { d_decimal_offsets = sizes_ptr; }

  /**
   * @brief Function that associates an existing stripe dictionary allocation
   */
  void attach_stripe_dict(gpu::StripeDictionary* host_stripe_dict,
                          gpu::StripeDictionary* dev_stripe_dict)
  {
    stripe_dict   = host_stripe_dict;
    d_stripe_dict = dev_stripe_dict;
  }
  [[nodiscard]] auto host_stripe_dict(size_t stripe) const
  {
    CUDF_EXPECTS(is_string(), "Stripe dictionary is only present in string columns.");
    return &stripe_dict[stripe * _dict_stride + _str_idx];
  }
  [[nodiscard]] auto device_stripe_dict() const noexcept { return d_stripe_dict; }

  // Index in the table
  [[nodiscard]] uint32_t index() const noexcept { return _index; }
  // Id in the ORC file
  [[nodiscard]] auto id() const noexcept { return _index + 1; }

  [[nodiscard]] auto is_child() const noexcept { return _is_child; }
  auto parent_index() const noexcept { return _parent_index.value(); }
  auto child_begin() const noexcept { return children.cbegin(); }
  auto child_end() const noexcept { return children.cend(); }
  auto num_children() const noexcept { return children.size(); }

  [[nodiscard]] auto type_width() const noexcept { return _type_width; }
  auto size() const noexcept { return cudf_column.size(); }

  auto null_count() const noexcept { return cudf_column.null_count(); }
  auto null_mask() const noexcept { return cudf_column.null_mask(); }
  [[nodiscard]] bool nullable() const noexcept { return null_mask() != nullptr; }
  auto user_defined_nullable() const noexcept { return nullable_from_metadata; }

  [[nodiscard]] auto scale() const noexcept { return _scale; }
  [[nodiscard]] auto precision() const noexcept { return _precision; }

  void set_orc_encoding(ColumnEncodingKind e) noexcept { _encoding_kind = e; }
  [[nodiscard]] auto orc_kind() const noexcept { return _type_kind; }
  [[nodiscard]] auto orc_encoding() const noexcept { return _encoding_kind; }
  [[nodiscard]] std::string_view orc_name() const noexcept { return name; }

 private:
  column_view cudf_column;

  // Identifier within the set of columns
  uint32_t _index = 0;
  // Identifier within the set of string columns
  int _str_idx;
  bool _is_child = false;

  // ORC-related members
  TypeKind _type_kind               = INVALID_TYPE_KIND;
  ColumnEncodingKind _encoding_kind = INVALID_ENCODING_KIND;
  std::string name;

  size_t _type_width = 0;
  int32_t _scale     = 0;
  int32_t _precision = 0;

  // String dictionary-related members
  size_t _dict_stride                        = 0;
  gpu::DictionaryChunk const* dict           = nullptr;
  gpu::StripeDictionary const* stripe_dict   = nullptr;
  gpu::DictionaryChunk const* d_dict         = nullptr;
  gpu::StripeDictionary const* d_stripe_dict = nullptr;

  // Offsets for encoded decimal elements. Used to enable direct writing of encoded decimal elements
  // into the output stream.
  uint32_t* d_decimal_offsets = nullptr;

  std::optional<bool> nullable_from_metadata;
  std::vector<uint32_t> children;
  std::optional<uint32_t> _parent_index;
};

size_type orc_table_view::num_rows() const noexcept
{
  return columns.empty() ? 0 : columns.front().size();
}

/**
 * @brief Gathers stripe information.
 *
 * @param columns List of columns
 * @param rowgroup_bounds Ranges of rows in each rowgroup [rowgroup][column]
 * @param max_stripe_size Maximum size of each stripe, both in bytes and in rows
 * @return List of stripe descriptors
 */
file_segmentation calculate_segmentation(host_span<orc_column_view const> columns,
                                         hostdevice_2dvector<rowgroup_rows>&& rowgroup_bounds,
                                         stripe_size_limits max_stripe_size)
{
  std::vector<stripe_rowgroups> infos;
  auto const num_rowgroups = rowgroup_bounds.size().first;
  size_t stripe_start      = 0;
  size_t stripe_bytes      = 0;
  size_type stripe_rows    = 0;
  for (size_t rg_idx = 0; rg_idx < num_rowgroups; ++rg_idx) {
    auto const rowgroup_total_bytes =
      std::accumulate(columns.begin(), columns.end(), 0ul, [&](size_t total_size, auto const& col) {
        auto const rows = rowgroup_bounds[rg_idx][col.index()].size();
        if (col.is_string()) {
          const auto dt = col.host_dict_chunk(rg_idx);
          return total_size + rows + dt->string_char_count;
        } else {
          return total_size + col.type_width() * rows;
        }
      });

    auto const rowgroup_rows_max =
      std::max_element(rowgroup_bounds[rg_idx].begin(),
                       rowgroup_bounds[rg_idx].end(),
                       [](auto& l, auto& r) { return l.size() < r.size(); })
        ->size();
    // Check if adding the current rowgroup to the stripe will make the stripe too large or long
    if ((rg_idx > stripe_start) && (stripe_bytes + rowgroup_total_bytes > max_stripe_size.bytes ||
                                    stripe_rows + rowgroup_rows_max > max_stripe_size.rows)) {
      infos.emplace_back(infos.size(), stripe_start, rg_idx - stripe_start);
      stripe_start = rg_idx;
      stripe_bytes = 0;
      stripe_rows  = 0;
    }

    stripe_bytes += rowgroup_total_bytes;
    stripe_rows += rowgroup_rows_max;
    if (rg_idx + 1 == num_rowgroups) {
      infos.emplace_back(infos.size(), stripe_start, num_rowgroups - stripe_start);
    }
  }

  return {std::move(rowgroup_bounds), std::move(infos)};
}

/**
 * @brief Builds up column dictionaries indices
 *
 * @param orc_table Non-owning view of a cuDF table w/ ORC-related info
 * @param rowgroup_bounds Ranges of rows in each rowgroup [rowgroup][column]
 * @param dict_data Dictionary data memory
 * @param dict_index Dictionary index memory
 * @param dict List of dictionary chunks
 * @param stream CUDA stream used for device memory operations and kernel launches
 */
void init_dictionaries(orc_table_view& orc_table,
                       device_2dspan<rowgroup_rows const> rowgroup_bounds,
                       device_span<device_span<uint32_t>> dict_data,
                       device_span<device_span<uint32_t>> dict_index,
                       hostdevice_2dvector<gpu::DictionaryChunk>* dict,
                       rmm::cuda_stream_view stream)
{
  // Setup per-rowgroup dictionary indexes for each dictionary-aware column
  for (auto col_idx : orc_table.string_column_indices) {
    auto& str_column = orc_table.column(col_idx);
    str_column.set_dict_stride(orc_table.num_string_columns());
    str_column.attach_dict_chunk(dict->base_host_ptr(), dict->base_device_ptr());
  }

  // Allocate temporary memory for dictionary indices
  std::vector<rmm::device_uvector<uint32_t>> dict_indices;
  dict_indices.reserve(orc_table.num_string_columns());
  std::transform(orc_table.string_column_indices.cbegin(),
                 orc_table.string_column_indices.cend(),
                 std::back_inserter(dict_indices),
                 [&](auto& col_idx) {
                   auto& str_column = orc_table.column(col_idx);
                   return cudf::detail::make_zeroed_device_uvector_async<uint32_t>(
                     str_column.size(), stream);
                 });

  // Create views of the temporary buffers in device memory
  std::vector<device_span<uint32_t>> dict_indices_views;
  dict_indices_views.reserve(dict_indices.size());
  std::transform(
    dict_indices.begin(), dict_indices.end(), std::back_inserter(dict_indices_views), [](auto& di) {
      return device_span<uint32_t>{di};
    });
  auto d_dict_indices_views = cudf::detail::make_device_uvector_async(dict_indices_views, stream);

  gpu::InitDictionaryIndices(orc_table.d_columns,
                             *dict,
                             dict_data,
                             dict_index,
                             d_dict_indices_views,
                             rowgroup_bounds,
                             orc_table.d_string_column_indices,
                             stream);
  dict->device_to_host(stream, true);
}

void writer::impl::build_dictionaries(orc_table_view& orc_table,
                                      host_span<stripe_rowgroups const> stripe_bounds,
                                      hostdevice_2dvector<gpu::DictionaryChunk> const& dict,
                                      host_span<rmm::device_uvector<uint32_t>> dict_index,
                                      host_span<bool const> dictionary_enabled,
                                      hostdevice_2dvector<gpu::StripeDictionary>& stripe_dict)
{
  const auto num_rowgroups = dict.size().first;

  for (size_t dict_idx = 0; dict_idx < orc_table.num_string_columns(); ++dict_idx) {
    auto& str_column = orc_table.string_column(dict_idx);
    str_column.attach_stripe_dict(stripe_dict.base_host_ptr(), stripe_dict.base_device_ptr());

    for (auto const& stripe : stripe_bounds) {
      auto& sd           = stripe_dict[stripe.id][dict_idx];
      sd.dict_data       = str_column.host_dict_chunk(stripe.first)->dict_data;
      sd.dict_index      = dict_index[dict_idx].data();  // Indexed by abs row
      sd.column_id       = orc_table.string_column_indices[dict_idx];
      sd.start_chunk     = stripe.first;
      sd.num_chunks      = stripe.size;
      sd.dict_char_count = 0;
      sd.num_strings =
        std::accumulate(stripe.cbegin(), stripe.cend(), 0, [&](auto dt_str_cnt, auto rg_idx) {
          const auto& dt = dict[rg_idx][dict_idx];
          return dt_str_cnt + dt.num_dict_strings;
        });
      sd.leaf_column = dict[0][dict_idx].leaf_column;
    }

    if (enable_dictionary_) {
      struct string_column_cost {
        size_t direct     = 0;
        size_t dictionary = 0;
      };
      auto const col_cost =
        std::accumulate(stripe_bounds.front().cbegin(),
                        stripe_bounds.back().cend(),
                        string_column_cost{},
                        [&](auto cost, auto rg_idx) -> string_column_cost {
                          const auto& dt = dict[rg_idx][dict_idx];
                          return {cost.direct + dt.string_char_count,
                                  cost.dictionary + dt.dict_char_count + dt.num_dict_strings};
                        });
      // Disable dictionary if it does not reduce the output size
      if (!dictionary_enabled[orc_table.string_column(dict_idx).index()] ||
          col_cost.dictionary >= col_cost.direct) {
        for (auto const& stripe : stripe_bounds) {
          stripe_dict[stripe.id][dict_idx].dict_data = nullptr;
        }
      }
    }
  }

  stripe_dict.host_to_device(stream);
  gpu::BuildStripeDictionaries(stripe_dict, stripe_dict, dict, stream);
  stripe_dict.device_to_host(stream, true);
}

/**
 * @brief Returns the maximum size of RLE encoded values of an integer type.
 **/
template <typename T>
size_t max_varint_size()
{
  // varint encodes 7 bits in each byte
  return cudf::util::div_rounding_up_unsafe(sizeof(T) * 8, 7);
}

constexpr size_t RLE_stream_size(TypeKind kind, size_t count)
{
  using cudf::util::div_rounding_up_unsafe;
  constexpr auto byte_rle_max_len = 128;
  switch (kind) {
    case TypeKind::BOOLEAN:
      return div_rounding_up_unsafe(count, byte_rle_max_len * 8) * (byte_rle_max_len + 1);
    case TypeKind::BYTE:
      return div_rounding_up_unsafe(count, byte_rle_max_len) * (byte_rle_max_len + 1);
    case TypeKind::SHORT:
      return div_rounding_up_unsafe(count, gpu::encode_block_size) *
             (gpu::encode_block_size * max_varint_size<int16_t>() + 2);
    case TypeKind::FLOAT:
    case TypeKind::INT:
    case TypeKind::DATE:
      return div_rounding_up_unsafe(count, gpu::encode_block_size) *
             (gpu::encode_block_size * max_varint_size<int32_t>() + 2);
    case TypeKind::LONG:
    case TypeKind::DOUBLE:
      return div_rounding_up_unsafe(count, gpu::encode_block_size) *
             (gpu::encode_block_size * max_varint_size<int64_t>() + 2);
    default: CUDF_FAIL("Unsupported ORC type for RLE stream size");
  }
}

auto uncomp_block_alignment(CompressionKind compression_kind)
{
  if (compression_kind == NONE or
      nvcomp::is_compression_disabled(to_nvcomp_compression_type(compression_kind))) {
    return 1u;
  }

  return 1u << nvcomp::compress_input_alignment_bits(to_nvcomp_compression_type(compression_kind));
}

auto comp_block_alignment(CompressionKind compression_kind)
{
  if (compression_kind == NONE or
      nvcomp::is_compression_disabled(to_nvcomp_compression_type(compression_kind))) {
    return 1u;
  }

  return 1u << nvcomp::compress_output_alignment_bits(to_nvcomp_compression_type(compression_kind));
}

orc_streams writer::impl::create_streams(host_span<orc_column_view> columns,
                                         file_segmentation const& segmentation,
                                         std::map<uint32_t, size_t> const& decimal_column_sizes)
{
  // 'column 0' row index stream
  std::vector<Stream> streams{{ROW_INDEX, 0}};  // TODO: Separate index and data streams?
  // First n + 1 streams are row index streams
  streams.reserve(columns.size() + 1);
  std::transform(columns.begin(), columns.end(), std::back_inserter(streams), [](auto const& col) {
    return Stream{ROW_INDEX, col.id()};
  });

  std::vector<int32_t> ids(columns.size() * gpu::CI_NUM_STREAMS, -1);
  std::vector<TypeKind> types(streams.size(), INVALID_TYPE_KIND);

  for (auto& column : columns) {
    auto const is_nullable = [&]() -> bool {
      if (single_write_mode) {
        return column.nullable();
      } else {
        // For chunked write, when not provided nullability, we assume the worst case scenario
        // that all columns are nullable.
        auto const chunked_nullable = column.user_defined_nullable().value_or(true);
        CUDF_EXPECTS(chunked_nullable or !column.nullable(),
                     "Mismatch in metadata prescribed nullability and input column nullability. "
                     "Metadata for nullable input column cannot prescribe nullability = false");
        return chunked_nullable;
      }
    }();

    auto RLE_column_size = [&](TypeKind type_kind) {
      return std::accumulate(
        thrust::make_counting_iterator(0ul),
        thrust::make_counting_iterator(segmentation.num_rowgroups()),
        0ul,
        [&](auto data_size, auto rg_idx) {
          return data_size +
                 RLE_stream_size(type_kind, segmentation.rowgroups[rg_idx][column.index()].size());
        });
    };

    auto const kind = column.orc_kind();

    auto add_stream =
      [&](gpu::StreamIndexType index_type, StreamKind kind, TypeKind type_kind, size_t size) {
        auto const max_alignment_padding = uncomp_block_alignment(compression_kind_) - 1;
        const auto base                  = column.index() * gpu::CI_NUM_STREAMS;
        ids[base + index_type]           = streams.size();
        streams.push_back(orc::Stream{
          kind,
          column.id(),
          (size == 0) ? 0 : size + max_alignment_padding * segmentation.num_rowgroups()});
        types.push_back(type_kind);
      };

    auto add_RLE_stream = [&](
                            gpu::StreamIndexType index_type, StreamKind kind, TypeKind type_kind) {
      add_stream(index_type, kind, type_kind, RLE_column_size(type_kind));
    };

    if (is_nullable) { add_RLE_stream(gpu::CI_PRESENT, PRESENT, TypeKind::BOOLEAN); }
    switch (kind) {
      case TypeKind::BOOLEAN:
      case TypeKind::BYTE:
        add_RLE_stream(gpu::CI_DATA, DATA, kind);
        column.set_orc_encoding(DIRECT);
        break;
      case TypeKind::SHORT:
      case TypeKind::INT:
      case TypeKind::LONG:
      case TypeKind::DATE:
        add_RLE_stream(gpu::CI_DATA, DATA, kind);
        column.set_orc_encoding(DIRECT_V2);
        break;
      case TypeKind::FLOAT:
      case TypeKind::DOUBLE:
        // Pass through if no nulls (no RLE encoding for floating point)
        add_stream(
          gpu::CI_DATA, DATA, kind, (column.null_count() != 0) ? RLE_column_size(kind) : 0);
        column.set_orc_encoding(DIRECT);
        break;
      case TypeKind::STRING: {
        bool enable_dict           = enable_dictionary_;
        size_t dict_data_size      = 0;
        size_t dict_strings        = 0;
        size_t dict_lengths_div512 = 0;
        for (auto const& stripe : segmentation.stripes) {
          const auto sd = column.host_stripe_dict(stripe.id);
          enable_dict   = (enable_dict && sd->dict_data != nullptr);
          if (enable_dict) {
            dict_strings += sd->num_strings;
            dict_lengths_div512 += (sd->num_strings + 0x1ff) >> 9;
            dict_data_size += sd->dict_char_count;
          }
        }

        auto const direct_data_size =
          segmentation.num_stripes() == 0
            ? 0
            : std::accumulate(segmentation.stripes.front().cbegin(),
                              segmentation.stripes.back().cend(),
                              size_t{0},
                              [&](auto data_size, auto rg_idx) {
                                return data_size +
                                       column.host_dict_chunk(rg_idx)->string_char_count;
                              });
        if (enable_dict) {
          uint32_t dict_bits = 0;
          for (dict_bits = 1; dict_bits < 32; dict_bits <<= 1) {
            if (dict_strings <= (1ull << dict_bits)) break;
          }
          const auto valid_count = column.size() - column.null_count();
          dict_data_size += (dict_bits * valid_count + 7) >> 3;
        }

        // Decide between direct or dictionary encoding
        if (enable_dict && dict_data_size < direct_data_size) {
          add_RLE_stream(gpu::CI_DATA, DATA, TypeKind::INT);
          add_stream(gpu::CI_DATA2, LENGTH, TypeKind::INT, dict_lengths_div512 * (512 * 4 + 2));
          add_stream(
            gpu::CI_DICTIONARY, DICTIONARY_DATA, TypeKind::CHAR, std::max(dict_data_size, 1ul));
          column.set_orc_encoding(DICTIONARY_V2);
        } else {
          add_stream(gpu::CI_DATA, DATA, TypeKind::CHAR, std::max<size_t>(direct_data_size, 1));
          add_RLE_stream(gpu::CI_DATA2, LENGTH, TypeKind::INT);
          column.set_orc_encoding(DIRECT_V2);
        }
        break;
      }
      case TypeKind::TIMESTAMP:
        add_RLE_stream(gpu::CI_DATA, DATA, TypeKind::LONG);
        add_RLE_stream(gpu::CI_DATA2, SECONDARY, TypeKind::LONG);
        column.set_orc_encoding(DIRECT_V2);
        break;
      case TypeKind::DECIMAL:
        // varint values (NO RLE)
        // data_stream_size = decimal_column_sizes.at(column.index());
        add_stream(gpu::CI_DATA, DATA, TypeKind::DECIMAL, decimal_column_sizes.at(column.index()));
        // scale stream TODO: compute exact size since all elems are equal
        add_RLE_stream(gpu::CI_DATA2, SECONDARY, TypeKind::INT);
        column.set_orc_encoding(DIRECT_V2);
        break;
      case TypeKind::LIST:
      case TypeKind::MAP:
        // no data stream, only lengths
        add_RLE_stream(gpu::CI_DATA2, LENGTH, TypeKind::INT);
        column.set_orc_encoding(DIRECT_V2);
        break;
      case TypeKind::STRUCT:
        // Only has the present stream
        break;
      default: CUDF_FAIL("Unsupported ORC type kind");
    }
  }
  return {std::move(streams), std::move(ids), std::move(types)};
}

orc_streams::orc_stream_offsets orc_streams::compute_offsets(
  host_span<orc_column_view const> columns, size_t num_rowgroups) const
{
  std::vector<size_t> strm_offsets(streams.size());
  size_t non_rle_data_size = 0;
  size_t rle_data_size     = 0;
  for (size_t i = 0; i < streams.size(); ++i) {
    const auto& stream = streams[i];

    auto const is_rle_data = [&]() {
      // First stream is an index stream, don't check types, etc.
      if (!stream.column_index().has_value()) return true;

      auto const& column = columns[stream.column_index().value()];
      // Dictionary encoded string column - dictionary characters or
      // directly encoded string - column characters
      if (column.orc_kind() == TypeKind::STRING &&
          ((stream.kind == DICTIONARY_DATA && column.orc_encoding() == DICTIONARY_V2) ||
           (stream.kind == DATA && column.orc_encoding() == DIRECT_V2)))
        return false;
      // Decimal data
      if (column.orc_kind() == TypeKind::DECIMAL && stream.kind == DATA) return false;

      // Everything else uses RLE
      return true;
    }();
    // non-RLE and RLE streams are separated in the buffer that stores encoded data
    // The computed offsets do not take the streams of the other type into account
    if (is_rle_data) {
      strm_offsets[i] = rle_data_size;
      rle_data_size += (stream.length + 7) & ~7;
    } else {
      strm_offsets[i] = non_rle_data_size;
      non_rle_data_size += stream.length;
    }
  }
  non_rle_data_size = (non_rle_data_size + 7) & ~7;

  return {std::move(strm_offsets), non_rle_data_size, rle_data_size};
}

std::vector<std::vector<rowgroup_rows>> calculate_aligned_rowgroup_bounds(
  orc_table_view const& orc_table,
  file_segmentation const& segmentation,
  rmm::cuda_stream_view stream)
{
  if (segmentation.num_rowgroups() == 0) return {};

  auto d_pd_set_counts_data = rmm::device_uvector<cudf::size_type>(
    orc_table.num_columns() * segmentation.num_rowgroups(), stream);
  auto const d_pd_set_counts = device_2dspan<cudf::size_type>{
    d_pd_set_counts_data.data(), segmentation.num_rowgroups(), orc_table.num_columns()};
  gpu::reduce_pushdown_masks(orc_table.d_columns, segmentation.rowgroups, d_pd_set_counts, stream);

  auto aligned_rgs = hostdevice_2dvector<rowgroup_rows>(
    segmentation.num_rowgroups(), orc_table.num_columns(), stream);
  CUDF_CUDA_TRY(hipMemcpyAsync(aligned_rgs.base_device_ptr(),
                                segmentation.rowgroups.base_device_ptr(),
                                aligned_rgs.count() * sizeof(rowgroup_rows),
                                hipMemcpyDefault,
                                stream.value()));
  auto const d_stripes = cudf::detail::make_device_uvector_async(segmentation.stripes, stream);

  // One thread per column, per stripe
  thrust::for_each_n(
    rmm::exec_policy(stream),
    thrust::make_counting_iterator(0),
    orc_table.num_columns() * segmentation.num_stripes(),
    [columns = device_span<orc_column_device_view const>{orc_table.d_columns},
     stripes = device_span<stripe_rowgroups const>{d_stripes},
     d_pd_set_counts,
     out_rowgroups = device_2dspan<rowgroup_rows>{aligned_rgs}] __device__(auto& idx) {
      uint32_t const col_idx = idx / stripes.size();
      // No alignment needed for root columns
      if (not columns[col_idx].parent_index.has_value()) return;

      auto const stripe_idx     = idx % stripes.size();
      auto const stripe         = stripes[stripe_idx];
      auto const parent_col_idx = columns[col_idx].parent_index.value();
      auto const parent_column  = columns[parent_col_idx];
      auto const stripe_end     = stripe.first + stripe.size;

      auto seek_last_borrow_rg = [&](auto rg_idx, size_type& bits_to_borrow) {
        auto curr         = rg_idx + 1;
        auto curr_rg_size = [&]() {
          return parent_column.pushdown_mask != nullptr ? d_pd_set_counts[curr][parent_col_idx]
                                                        : out_rowgroups[curr][col_idx].size();
        };
        while (curr < stripe_end and curr_rg_size() <= bits_to_borrow) {
          // All bits from rowgroup borrowed, make the rowgroup empty
          out_rowgroups[curr][col_idx].begin = out_rowgroups[curr][col_idx].end;
          bits_to_borrow -= curr_rg_size();
          ++curr;
        }
        return curr;
      };

      int previously_borrowed = 0;
      for (auto rg_idx = stripe.first; rg_idx + 1 < stripe_end; ++rg_idx) {
        auto& rg = out_rowgroups[rg_idx][col_idx];

        if (parent_column.pushdown_mask == nullptr) {
          // No pushdown mask, all null mask bits will be encoded
          // Align on rowgroup size (can be misaligned for list children)
          if (rg.size() % 8) {
            auto bits_to_borrow           = 8 - rg.size() % 8;
            auto const last_borrow_rg_idx = seek_last_borrow_rg(rg_idx, bits_to_borrow);
            if (last_borrow_rg_idx == stripe_end) {
              // Didn't find enough bits to borrow, move the rowgroup end to the stripe end
              rg.end = out_rowgroups[stripe_end - 1][col_idx].end;
              // Done with this stripe
              break;
            }
            auto& last_borrow_rg = out_rowgroups[last_borrow_rg_idx][col_idx];
            last_borrow_rg.begin += bits_to_borrow;
            rg.end = last_borrow_rg.begin;
            // Skip the rowgroups we emptied in the loop
            rg_idx = last_borrow_rg_idx - 1;
          }
        } else {
          // pushdown mask present; null mask bits w/ set pushdown mask bits will be encoded
          // Use the number of set bits in pushdown mask as size
          auto bits_to_borrow =
            8 - (d_pd_set_counts[rg_idx][parent_col_idx] - previously_borrowed) % 8;
          if (bits_to_borrow == 0) {
            // Didn't borrow any bits for this rowgroup
            previously_borrowed = 0;
            continue;
          }

          // Find rowgroup in which we finish the search for missing bits
          auto const last_borrow_rg_idx = seek_last_borrow_rg(rg_idx, bits_to_borrow);
          if (last_borrow_rg_idx == stripe_end) {
            // Didn't find enough bits to borrow, move the rowgroup end to the stripe end
            rg.end = out_rowgroups[stripe_end - 1][col_idx].end;
            // Done with this stripe
            break;
          }

          auto& last_borrow_rg = out_rowgroups[last_borrow_rg_idx][col_idx];
          // First row that does not need to be borrowed
          auto borrow_end = last_borrow_rg.begin;

          // Adjust the number of bits to borrow in the next iteration
          previously_borrowed = bits_to_borrow;

          // Find word in which we finish the search for missing bits (guaranteed to be available)
          while (bits_to_borrow != 0) {
            auto const mask = cudf::detail::get_mask_offset_word(
              parent_column.pushdown_mask, 0, borrow_end, borrow_end + 32);
            auto const valid_in_word = __popc(mask);

            if (valid_in_word > bits_to_borrow) break;
            bits_to_borrow -= valid_in_word;
            borrow_end += 32;
          }

          // Find the last of the missing bits (guaranteed to be available)
          while (bits_to_borrow != 0) {
            if (bit_is_set(parent_column.pushdown_mask, borrow_end)) { --bits_to_borrow; };
            ++borrow_end;
          }

          last_borrow_rg.begin = borrow_end;
          rg.end               = borrow_end;
          // Skip the rowgroups we emptied in the loop
          rg_idx = last_borrow_rg_idx - 1;
        }
      }
    });

  aligned_rgs.device_to_host(stream, true);

  std::vector<std::vector<rowgroup_rows>> h_aligned_rgs;
  h_aligned_rgs.reserve(segmentation.num_rowgroups());
  std::transform(thrust::make_counting_iterator(0ul),
                 thrust::make_counting_iterator(segmentation.num_rowgroups()),
                 std::back_inserter(h_aligned_rgs),
                 [&](auto idx) -> std::vector<rowgroup_rows> {
                   return {aligned_rgs[idx].begin(), aligned_rgs[idx].end()};
                 });

  return h_aligned_rgs;
}

struct segmented_valid_cnt_input {
  bitmask_type const* mask;
  std::vector<size_type> indices;
};

encoded_data encode_columns(orc_table_view const& orc_table,
                            string_dictionaries&& dictionaries,
                            encoder_decimal_info&& dec_chunk_sizes,
                            file_segmentation const& segmentation,
                            orc_streams const& streams,
                            uint32_t uncomp_block_align,
                            rmm::cuda_stream_view stream)
{
  auto const num_columns = orc_table.num_columns();
  hostdevice_2dvector<gpu::EncChunk> chunks(num_columns, segmentation.num_rowgroups(), stream);
  auto const stream_offsets =
    streams.compute_offsets(orc_table.columns, segmentation.num_rowgroups());
  rmm::device_uvector<uint8_t> encoded_data(stream_offsets.data_size(), stream);

  auto const aligned_rowgroups = calculate_aligned_rowgroup_bounds(orc_table, segmentation, stream);

  // Initialize column chunks' descriptions
  std::map<size_type, segmented_valid_cnt_input> validity_check_inputs;

  for (auto const& column : orc_table.columns) {
    for (auto const& stripe : segmentation.stripes) {
      for (auto rg_idx_it = stripe.cbegin(); rg_idx_it < stripe.cend(); ++rg_idx_it) {
        auto const rg_idx      = *rg_idx_it;
        auto& ck               = chunks[column.index()][rg_idx];
        ck.start_row           = segmentation.rowgroups[rg_idx][column.index()].begin;
        ck.num_rows            = segmentation.rowgroups[rg_idx][column.index()].size();
        ck.null_mask_start_row = aligned_rowgroups[rg_idx][column.index()].begin;
        ck.null_mask_num_rows  = aligned_rowgroups[rg_idx][column.index()].size();
        ck.encoding_kind       = column.orc_encoding();
        ck.type_kind           = column.orc_kind();
        if (ck.type_kind == TypeKind::STRING) {
          ck.dict_index = (ck.encoding_kind == DICTIONARY_V2)
                            ? column.host_stripe_dict(stripe.id)->dict_index
                            : nullptr;
          ck.dtype_len  = 1;
        } else {
          ck.dtype_len = column.type_width();
        }
        ck.scale = column.scale();
        if (ck.type_kind == TypeKind::DECIMAL) { ck.decimal_offsets = column.decimal_offsets(); }
      }
    }
  }
  chunks.host_to_device(stream);
  // TODO (future): pass columns separately from chunks (to skip this step)
  // and remove info from chunks that is common for the entire column
  thrust::for_each_n(
    rmm::exec_policy(stream),
    thrust::make_counting_iterator(0ul),
    chunks.count(),
    [chunks = device_2dspan<gpu::EncChunk>{chunks},
     cols = device_span<orc_column_device_view const>{orc_table.d_columns}] __device__(auto& idx) {
      auto const col_idx             = idx / chunks.size().second;
      auto const rg_idx              = idx % chunks.size().second;
      chunks[col_idx][rg_idx].column = &cols[col_idx];
    });

  auto validity_check_indices = [&](size_t col_idx) {
    std::vector<size_type> indices;
    for (auto const& stripe : segmentation.stripes) {
      for (auto rg_idx_it = stripe.cbegin(); rg_idx_it < stripe.cend() - 1; ++rg_idx_it) {
        auto const& chunk = chunks[col_idx][*rg_idx_it];
        indices.push_back(chunk.start_row);
        indices.push_back(chunk.start_row + chunk.num_rows);
      }
    }
    return indices;
  };
  for (auto const& column : orc_table.columns) {
    if (column.orc_kind() == TypeKind::BOOLEAN && column.nullable()) {
      validity_check_inputs[column.index()] = {column.null_mask(),
                                               validity_check_indices(column.index())};
    }
  }
  for (auto& cnt_in : validity_check_inputs) {
    auto const valid_counts =
      cudf::detail::segmented_valid_count(cnt_in.second.mask, cnt_in.second.indices, stream);
    CUDF_EXPECTS(
      std::none_of(valid_counts.cbegin(),
                   valid_counts.cend(),
                   [](auto valid_count) { return valid_count % 8; }),
      "There's currently a bug in encoding boolean columns. Suggested workaround is to convert "
      "to int8 type."
      " Please see https://github.com/rapidsai/cudf/issues/6763 for more information.");
  }

  hostdevice_2dvector<gpu::encoder_chunk_streams> chunk_streams(
    num_columns, segmentation.num_rowgroups(), stream);
  for (size_t col_idx = 0; col_idx < num_columns; col_idx++) {
    auto const& column = orc_table.column(col_idx);
    auto col_streams   = chunk_streams[col_idx];
    for (auto const& stripe : segmentation.stripes) {
      for (auto rg_idx_it = stripe.cbegin(); rg_idx_it < stripe.cend(); ++rg_idx_it) {
        auto const rg_idx = *rg_idx_it;
        auto const& ck    = chunks[col_idx][rg_idx];
        auto& strm        = col_streams[rg_idx];

        for (int strm_type = 0; strm_type < gpu::CI_NUM_STREAMS; ++strm_type) {
          auto const strm_id = streams.id(col_idx * gpu::CI_NUM_STREAMS + strm_type);

          strm.ids[strm_type] = strm_id;
          if (strm_id >= 0) {
            if ((strm_type == gpu::CI_DICTIONARY) ||
                (strm_type == gpu::CI_DATA2 && ck.encoding_kind == DICTIONARY_V2)) {
              if (rg_idx_it == stripe.cbegin()) {
                const int32_t dict_stride = column.dict_stride();
                const auto stripe_dict    = column.host_stripe_dict(stripe.id);
                strm.lengths[strm_type] =
                  (strm_type == gpu::CI_DICTIONARY)
                    ? stripe_dict->dict_char_count
                    : (((stripe_dict->num_strings + 0x1ff) >> 9) * (512 * 4 + 2));
                if (stripe.id == 0) {
                  strm.data_ptrs[strm_type] = encoded_data.data() + stream_offsets.offsets[strm_id];
                  // Dictionary lengths are encoded as RLE, which are all stored after non-RLE data:
                  // include non-RLE data size in the offset only in that case
                  if (strm_type == gpu::CI_DATA2 && ck.encoding_kind == DICTIONARY_V2)
                    strm.data_ptrs[strm_type] += stream_offsets.non_rle_data_size;
                } else {
                  auto const& strm_up = col_streams[stripe_dict[-dict_stride].start_chunk];
                  strm.data_ptrs[strm_type] =
                    strm_up.data_ptrs[strm_type] + strm_up.lengths[strm_type];
                }
              } else {
                strm.lengths[strm_type]   = 0;
                strm.data_ptrs[strm_type] = col_streams[rg_idx - 1].data_ptrs[strm_type];
              }
            } else if (strm_type == gpu::CI_DATA && ck.type_kind == TypeKind::STRING &&
                       ck.encoding_kind == DIRECT_V2) {
              strm.lengths[strm_type]   = column.host_dict_chunk(rg_idx)->string_char_count;
              strm.data_ptrs[strm_type] = (rg_idx == 0)
                                            ? encoded_data.data() + stream_offsets.offsets[strm_id]
                                            : (col_streams[rg_idx - 1].data_ptrs[strm_type] +
                                               col_streams[rg_idx - 1].lengths[strm_type]);
            } else if (strm_type == gpu::CI_DATA && streams[strm_id].length == 0 &&
                       (ck.type_kind == DOUBLE || ck.type_kind == FLOAT)) {
              // Pass-through
              strm.lengths[strm_type]   = ck.num_rows * ck.dtype_len;
              strm.data_ptrs[strm_type] = nullptr;

            } else if (ck.type_kind == DECIMAL && strm_type == gpu::CI_DATA) {
              strm.lengths[strm_type]   = dec_chunk_sizes.rg_sizes.at(col_idx)[rg_idx];
              strm.data_ptrs[strm_type] = (rg_idx == 0)
                                            ? encoded_data.data() + stream_offsets.offsets[strm_id]
                                            : (col_streams[rg_idx - 1].data_ptrs[strm_type] +
                                               col_streams[rg_idx - 1].lengths[strm_type]);
            } else {
              strm.lengths[strm_type] = RLE_stream_size(streams.type(strm_id), ck.num_rows);
              // RLE encoded streams are stored after all non-RLE streams
              strm.data_ptrs[strm_type] =
                (rg_idx == 0) ? (encoded_data.data() + stream_offsets.non_rle_data_size +
                                 stream_offsets.offsets[strm_id])
                              : (col_streams[rg_idx - 1].data_ptrs[strm_type] +
                                 col_streams[rg_idx - 1].lengths[strm_type]);
            }
          } else {
            strm.lengths[strm_type]   = 0;
            strm.data_ptrs[strm_type] = nullptr;
          }
          auto const misalignment =
            reinterpret_cast<intptr_t>(strm.data_ptrs[strm_type]) % uncomp_block_align;
          if (misalignment != 0) {
            strm.data_ptrs[strm_type] += (uncomp_block_align - misalignment);
          }
        }
      }
    }
  }

  chunk_streams.host_to_device(stream);

  if (orc_table.num_rows() > 0) {
    if (orc_table.num_string_columns() != 0) {
      auto d_stripe_dict = orc_table.string_column(0).device_stripe_dict();
      gpu::EncodeStripeDictionaries(d_stripe_dict,
                                    chunks,
                                    orc_table.num_string_columns(),
                                    segmentation.num_stripes(),
                                    chunk_streams,
                                    stream);
    }

    gpu::EncodeOrcColumnData(chunks, chunk_streams, stream);
  }
  dictionaries.data.clear();
  dictionaries.index.clear();
  stream.synchronize();

  return {std::move(encoded_data), std::move(chunk_streams)};
}

std::vector<StripeInformation> writer::impl::gather_stripes(
  size_t num_index_streams,
  file_segmentation const& segmentation,
  hostdevice_2dvector<gpu::encoder_chunk_streams>* enc_streams,
  hostdevice_2dvector<gpu::StripeStream>* strm_desc)
{
  if (segmentation.num_stripes() == 0) { return {}; }
  std::vector<StripeInformation> stripes(segmentation.num_stripes());
  for (auto const& stripe : segmentation.stripes) {
    for (size_t col_idx = 0; col_idx < enc_streams->size().first; col_idx++) {
      const auto& strm = (*enc_streams)[col_idx][stripe.first];

      // Assign stream data of column data stream(s)
      for (int k = 0; k < gpu::CI_INDEX; k++) {
        const auto stream_id = strm.ids[k];
        if (stream_id != -1) {
          auto* ss           = &(*strm_desc)[stripe.id][stream_id - num_index_streams];
          ss->stream_size    = 0;
          ss->first_chunk_id = stripe.first;
          ss->num_chunks     = stripe.size;
          ss->column_id      = col_idx;
          ss->stream_type    = k;
        }
      }
    }

    stripes[stripe.id].numberOfRows =
      stripe.size == 0 ? 0
                       : segmentation.rowgroups[stripe.first + stripe.size - 1][0].end -
                           segmentation.rowgroups[stripe.first][0].begin;
  }

  strm_desc->host_to_device(stream);
  gpu::CompactOrcDataStreams(*strm_desc, *enc_streams, stream);
  strm_desc->device_to_host(stream);
  enc_streams->device_to_host(stream, true);

  return stripes;
}

void set_stat_desc_leaf_cols(device_span<orc_column_device_view const> columns,
                             device_span<stats_column_desc> stat_desc,
                             rmm::cuda_stream_view stream)
{
  thrust::for_each(rmm::exec_policy(stream),
                   thrust::make_counting_iterator(0ul),
                   thrust::make_counting_iterator(stat_desc.size()),
                   [=] __device__(auto idx) { stat_desc[idx].leaf_column = &columns[idx]; });
}

hostdevice_vector<uint8_t> allocate_and_encode_blobs(
  hostdevice_vector<statistics_merge_group>& stats_merge_groups,
  rmm::device_uvector<statistics_chunk>& stat_chunks,
  int num_stat_blobs,
  rmm::cuda_stream_view stream)
{
  // figure out the buffer size needed for protobuf format
  gpu::orc_init_statistics_buffersize(
    stats_merge_groups.device_ptr(), stat_chunks.data(), num_stat_blobs, stream);
  auto max_blobs = stats_merge_groups.element(num_stat_blobs - 1, stream);

  hostdevice_vector<uint8_t> blobs(max_blobs.start_chunk + max_blobs.num_chunks, stream);
  gpu::orc_encode_statistics(blobs.device_ptr(),
                             stats_merge_groups.device_ptr(),
                             stat_chunks.data(),
                             num_stat_blobs,
                             stream);
  stats_merge_groups.device_to_host(stream);
  blobs.device_to_host(stream, true);
  return blobs;
}

writer::impl::intermediate_statistics writer::impl::gather_statistic_blobs(
  statistics_freq const stats_freq,
  orc_table_view const& orc_table,
  file_segmentation const& segmentation)
{
  auto const num_rowgroup_blobs     = segmentation.rowgroups.count();
  auto const num_stripe_blobs       = segmentation.num_stripes() * orc_table.num_columns();
  auto const are_statistics_enabled = stats_freq != statistics_freq::STATISTICS_NONE;
  if (not are_statistics_enabled or num_rowgroup_blobs + num_stripe_blobs == 0) {
    return writer::impl::intermediate_statistics{stream};
  }

  hostdevice_vector<stats_column_desc> stat_desc(orc_table.num_columns(), stream);
  hostdevice_vector<statistics_merge_group> rowgroup_merge(num_rowgroup_blobs, stream);
  hostdevice_vector<statistics_merge_group> stripe_merge(num_stripe_blobs, stream);
  std::vector<statistics_dtype> col_stats_dtypes;
  std::vector<data_type> col_types;
  auto rowgroup_stat_merge = rowgroup_merge.host_ptr();
  auto stripe_stat_merge   = stripe_merge.host_ptr();

  for (auto const& column : orc_table.columns) {
    stats_column_desc* desc = &stat_desc[column.index()];
    switch (column.orc_kind()) {
      case TypeKind::BYTE: desc->stats_dtype = dtype_int8; break;
      case TypeKind::SHORT: desc->stats_dtype = dtype_int16; break;
      case TypeKind::INT: desc->stats_dtype = dtype_int32; break;
      case TypeKind::LONG: desc->stats_dtype = dtype_int64; break;
      case TypeKind::FLOAT: desc->stats_dtype = dtype_float32; break;
      case TypeKind::DOUBLE: desc->stats_dtype = dtype_float64; break;
      case TypeKind::BOOLEAN: desc->stats_dtype = dtype_bool; break;
      case TypeKind::DATE: desc->stats_dtype = dtype_int32; break;
      case TypeKind::DECIMAL: desc->stats_dtype = dtype_decimal64; break;
      case TypeKind::TIMESTAMP: desc->stats_dtype = dtype_timestamp64; break;
      case TypeKind::STRING: desc->stats_dtype = dtype_string; break;
      default: desc->stats_dtype = dtype_none; break;
    }
    desc->num_rows   = column.size();
    desc->num_values = column.size();
    if (desc->stats_dtype == dtype_timestamp64) {
      // Timestamp statistics are in milliseconds
      switch (column.scale()) {
        case 9: desc->ts_scale = 1000; break;
        case 6: desc->ts_scale = 0; break;
        case 3: desc->ts_scale = -1000; break;
        case 0: desc->ts_scale = -1000000; break;
        default: desc->ts_scale = 0; break;
      }
    } else {
      desc->ts_scale = 0;
    }
    col_stats_dtypes.push_back(desc->stats_dtype);
    col_types.push_back(column.type());
    for (auto const& stripe : segmentation.stripes) {
      auto& grp       = stripe_stat_merge[column.index() * segmentation.num_stripes() + stripe.id];
      grp.col_dtype   = column.type();
      grp.stats_dtype = desc->stats_dtype;
      grp.start_chunk =
        static_cast<uint32_t>(column.index() * segmentation.num_rowgroups() + stripe.first);
      grp.num_chunks = stripe.size;
      for (auto rg_idx_it = stripe.cbegin(); rg_idx_it != stripe.cend(); ++rg_idx_it) {
        auto& rg_grp =
          rowgroup_stat_merge[column.index() * segmentation.num_rowgroups() + *rg_idx_it];
        rg_grp.col_dtype   = column.type();
        rg_grp.stats_dtype = desc->stats_dtype;
        rg_grp.start_chunk = *rg_idx_it;
        rg_grp.num_chunks  = 1;
      }
    }
  }
  stat_desc.host_to_device(stream);
  rowgroup_merge.host_to_device(stream);
  stripe_merge.host_to_device(stream);
  set_stat_desc_leaf_cols(orc_table.d_columns, stat_desc, stream);

  // The rowgroup stat chunks are written out in each stripe. The stripe and file-level chunks are
  // written in the footer. To prevent persisting the rowgroup stat chunks across multiple write
  // calls in a chunked write situation, these allocations are split up so stripe data can persist
  // until the footer is written and rowgroup data can be freed after being written to the stripe.
  rmm::device_uvector<statistics_chunk> rowgroup_chunks(num_rowgroup_blobs, stream);
  rmm::device_uvector<statistics_chunk> stripe_chunks(num_stripe_blobs, stream);
  auto rowgroup_stat_chunks = rowgroup_chunks.data();
  auto stripe_stat_chunks   = stripe_chunks.data();

  rmm::device_uvector<statistics_group> rowgroup_groups(num_rowgroup_blobs, stream);
  gpu::orc_init_statistics_groups(
    rowgroup_groups.data(), stat_desc.device_ptr(), segmentation.rowgroups, stream);

  detail::calculate_group_statistics<detail::io_file_format::ORC>(
    rowgroup_chunks.data(), rowgroup_groups.data(), num_rowgroup_blobs, stream);

  detail::merge_group_statistics<detail::io_file_format::ORC>(
    stripe_stat_chunks, rowgroup_stat_chunks, stripe_merge.device_ptr(), num_stripe_blobs, stream);

  // With chunked writes, the orc table can be deallocated between write calls.
  // This forces our hand to encode row groups and stripes only in this stage and further
  // we have to persist any data from the table that we need later. The
  // minimum and maximum string inside the `str_val` structure inside `statistics_val` in
  // `statistic_chunk` that are copies of the largest and smallest strings in the row group,
  // or stripe need to be persisted between write calls. We write rowgroup data with each
  // stripe and then save each stripe's stats until the end where we merge those all together
  // to get the file-level stats.

  // Skip rowgroup blobs when encoding, if chosen granularity is coarser than "ROW_GROUP".
  auto const is_granularity_rowgroup = stats_freq == ORC_STATISTICS_ROW_GROUP;
  // we have to encode the row groups now IF they are being written out
  auto rowgroup_blobs = [&]() -> std::vector<ColStatsBlob> {
    if (not is_granularity_rowgroup) { return {}; }

    hostdevice_vector<uint8_t> blobs =
      allocate_and_encode_blobs(rowgroup_merge, rowgroup_chunks, num_rowgroup_blobs, stream);

    std::vector<ColStatsBlob> rowgroup_blobs(num_rowgroup_blobs);
    for (size_t i = 0; i < num_rowgroup_blobs; i++) {
      auto const stat_begin = blobs.host_ptr(rowgroup_merge[i].start_chunk);
      auto const stat_end   = stat_begin + rowgroup_merge[i].num_chunks;
      rowgroup_blobs[i].assign(stat_begin, stat_end);
    }
    return rowgroup_blobs;
  }();

  return {std::move(rowgroup_blobs),
          std::move(stripe_chunks),
          std::move(stripe_merge),
          std::move(col_stats_dtypes),
          std::move(col_types)};
}

writer::impl::encoded_footer_statistics writer::impl::finish_statistic_blobs(
  int num_stripes, writer::impl::persisted_statistics& per_chunk_stats)
{
  auto stripe_size_iter = thrust::make_transform_iterator(per_chunk_stats.stripe_stat_merge.begin(),
                                                          [](auto const& i) { return i.size(); });

  auto const num_columns = per_chunk_stats.col_types.size();
  auto const num_stripe_blobs =
    thrust::reduce(stripe_size_iter, stripe_size_iter + per_chunk_stats.stripe_stat_merge.size());
  auto const num_file_blobs = num_columns;
  auto const num_blobs      = static_cast<int>(num_stripe_blobs + num_file_blobs);

  if (num_stripe_blobs == 0) { return {}; }

  // merge the stripe persisted data and add file data
  rmm::device_uvector<statistics_chunk> stat_chunks(num_blobs, stream);
  hostdevice_vector<statistics_merge_group> stats_merge(num_blobs, stream);

  // we need to merge the stat arrays from the persisted data.
  // this needs to be done carefully because each array can contain
  // a different number of stripes and stripes from each column must be
  // located next to each other. We know the total number of stripes and
  // we know the size of each array. The number of stripes per column in a chunk array can
  // be calculated by dividing the number of chunks by the number of columns.
  // That many chunks need to be copied at a time to the proper destination.
  size_t num_entries_seen = 0;
  for (size_t i = 0; i < per_chunk_stats.stripe_stat_chunks.size(); ++i) {
    auto const stripes_per_col = per_chunk_stats.stripe_stat_chunks[i].size() / num_columns;

    auto const chunk_bytes = stripes_per_col * sizeof(statistics_chunk);
    auto const merge_bytes = stripes_per_col * sizeof(statistics_merge_group);
    for (size_t col = 0; col < num_columns; ++col) {
      hipMemcpyAsync(stat_chunks.data() + (num_stripes * col) + num_entries_seen,
                      per_chunk_stats.stripe_stat_chunks[i].data() + col * stripes_per_col,
                      chunk_bytes,
                      hipMemcpyDeviceToDevice,
                      stream);
      hipMemcpyAsync(stats_merge.device_ptr() + (num_stripes * col) + num_entries_seen,
                      per_chunk_stats.stripe_stat_merge[i].device_ptr() + col * stripes_per_col,
                      merge_bytes,
                      hipMemcpyDeviceToDevice,
                      stream);
    }
    num_entries_seen += stripes_per_col;
  }

  std::vector<statistics_merge_group> file_stats_merge(num_file_blobs);
  for (auto i = 0u; i < num_file_blobs; ++i) {
    auto col_stats         = &file_stats_merge[i];
    col_stats->col_dtype   = per_chunk_stats.col_types[i];
    col_stats->stats_dtype = per_chunk_stats.stats_dtypes[i];
    col_stats->start_chunk = static_cast<uint32_t>(i * num_stripes);
    col_stats->num_chunks  = static_cast<uint32_t>(num_stripes);
  }

  auto d_file_stats_merge = stats_merge.device_ptr(num_stripe_blobs);
  hipMemcpyAsync(d_file_stats_merge,
                  file_stats_merge.data(),
                  num_file_blobs * sizeof(statistics_merge_group),
                  hipMemcpyHostToDevice,
                  stream);

  auto file_stat_chunks = stat_chunks.data() + num_stripe_blobs;
  detail::merge_group_statistics<detail::io_file_format::ORC>(
    file_stat_chunks, stat_chunks.data(), d_file_stats_merge, num_file_blobs, stream);

  hostdevice_vector<uint8_t> blobs =
    allocate_and_encode_blobs(stats_merge, stat_chunks, num_blobs, stream);

  auto stripe_stat_merge = stats_merge.host_ptr();

  std::vector<ColStatsBlob> stripe_blobs(num_stripe_blobs);
  for (size_t i = 0; i < num_stripe_blobs; i++) {
    auto const stat_begin = blobs.host_ptr(stripe_stat_merge[i].start_chunk);
    auto const stat_end   = stat_begin + stripe_stat_merge[i].num_chunks;
    stripe_blobs[i].assign(stat_begin, stat_end);
  }

  std::vector<ColStatsBlob> file_blobs(num_file_blobs);
  auto file_stat_merge = stats_merge.host_ptr(num_stripe_blobs);
  for (auto i = 0u; i < num_file_blobs; i++) {
    auto const stat_begin = blobs.host_ptr(file_stat_merge[i].start_chunk);
    auto const stat_end   = stat_begin + file_stat_merge[i].num_chunks;
    file_blobs[i].assign(stat_begin, stat_end);
  }

  return {std::move(stripe_blobs), std::move(file_blobs)};
}

void writer::impl::write_index_stream(int32_t stripe_id,
                                      int32_t stream_id,
                                      host_span<orc_column_view const> columns,
                                      file_segmentation const& segmentation,
                                      host_2dspan<gpu::encoder_chunk_streams const> enc_streams,
                                      host_2dspan<gpu::StripeStream const> strm_desc,
                                      host_span<compression_result const> comp_res,
                                      std::vector<ColStatsBlob> const& rg_stats,
                                      StripeInformation* stripe,
                                      orc_streams* streams,
                                      ProtobufWriter* pbw)
{
  row_group_index_info present;
  row_group_index_info data;
  row_group_index_info data2;
  auto const column_id = stream_id - 1;

  auto find_record = [=, &strm_desc](gpu::encoder_chunk_streams const& stream,
                                     gpu::StreamIndexType type) {
    row_group_index_info record;
    if (stream.ids[type] > 0) {
      record.pos = 0;
      if (compression_kind_ != NONE) {
        auto const& ss   = strm_desc[stripe_id][stream.ids[type] - (columns.size() + 1)];
        record.blk_pos   = ss.first_block;
        record.comp_pos  = 0;
        record.comp_size = ss.stream_size;
      }
    }
    return record;
  };
  auto scan_record = [=, &comp_res](gpu::encoder_chunk_streams const& stream,
                                    gpu::StreamIndexType type,
                                    row_group_index_info& record) {
    if (record.pos >= 0) {
      record.pos += stream.lengths[type];
      while ((record.pos >= 0) && (record.blk_pos >= 0) &&
             (static_cast<size_t>(record.pos) >= compression_blocksize_) &&
             (record.comp_pos + block_header_size + comp_res[record.blk_pos].bytes_written <
              static_cast<size_t>(record.comp_size))) {
        record.pos -= compression_blocksize_;
        record.comp_pos += block_header_size + comp_res[record.blk_pos].bytes_written;
        record.blk_pos += 1;
      }
    }
  };

  auto kind = TypeKind::STRUCT;
  // TBD: Not sure we need an empty index stream for column 0
  if (stream_id != 0) {
    const auto& strm = enc_streams[column_id][0];
    present          = find_record(strm, gpu::CI_PRESENT);
    data             = find_record(strm, gpu::CI_DATA);
    data2            = find_record(strm, gpu::CI_DATA2);

    // Change string dictionary to int from index point of view
    kind = columns[column_id].orc_kind();
    if (kind == TypeKind::STRING && columns[column_id].orc_encoding() == DICTIONARY_V2) {
      kind = TypeKind::INT;
    }
  }

  buffer_.resize((compression_kind_ != NONE) ? 3 : 0);

  // Add row index entries
  auto const& rowgroups_range = segmentation.stripes[stripe_id];
  std::for_each(rowgroups_range.cbegin(), rowgroups_range.cend(), [&](auto rowgroup) {
    pbw->put_row_index_entry(present.comp_pos,
                             present.pos,
                             data.comp_pos,
                             data.pos,
                             data2.comp_pos,
                             data2.pos,
                             kind,
                             (rg_stats.empty() or stream_id == 0)
                               ? nullptr
                               : (&rg_stats[column_id * segmentation.num_rowgroups() + rowgroup]));

    if (stream_id != 0) {
      const auto& strm = enc_streams[column_id][rowgroup];
      scan_record(strm, gpu::CI_PRESENT, present);
      scan_record(strm, gpu::CI_DATA, data);
      scan_record(strm, gpu::CI_DATA2, data2);
    }
  });

  (*streams)[stream_id].length = buffer_.size();
  if (compression_kind_ != NONE) {
    uint32_t uncomp_ix_len = (uint32_t)((*streams)[stream_id].length - 3) * 2 + 1;
    buffer_[0]             = static_cast<uint8_t>(uncomp_ix_len >> 0);
    buffer_[1]             = static_cast<uint8_t>(uncomp_ix_len >> 8);
    buffer_[2]             = static_cast<uint8_t>(uncomp_ix_len >> 16);
  }
  out_sink_->host_write(buffer_.data(), buffer_.size());
  stripe->indexLength += buffer_.size();
}

std::future<void> writer::impl::write_data_stream(gpu::StripeStream const& strm_desc,
                                                  gpu::encoder_chunk_streams const& enc_stream,
                                                  uint8_t const* compressed_data,
                                                  uint8_t* stream_out,
                                                  StripeInformation* stripe,
                                                  orc_streams* streams)
{
  const auto length                                        = strm_desc.stream_size;
  (*streams)[enc_stream.ids[strm_desc.stream_type]].length = length;
  if (length == 0) {
    return std::async(std::launch::deferred, [] {});
  }

  const auto* stream_in = (compression_kind_ == NONE) ? enc_stream.data_ptrs[strm_desc.stream_type]
                                                      : (compressed_data + strm_desc.bfr_offset);

  auto write_task = [&]() {
    if (out_sink_->is_device_write_preferred(length)) {
      return out_sink_->device_write_async(stream_in, length, stream);
    } else {
      CUDF_CUDA_TRY(
        hipMemcpyAsync(stream_out, stream_in, length, hipMemcpyDeviceToHost, stream.value()));
      stream.synchronize();

      out_sink_->host_write(stream_out, length);
      return std::async(std::launch::deferred, [] {});
    }
  }();
  stripe->dataLength += length;
  return write_task;
}

void writer::impl::add_uncompressed_block_headers(std::vector<uint8_t>& v)
{
  if (compression_kind_ != NONE) {
    size_t uncomp_len = v.size() - 3, pos = 0, block_len;
    while (uncomp_len > compression_blocksize_) {
      block_len  = compression_blocksize_ * 2 + 1;
      v[pos + 0] = static_cast<uint8_t>(block_len >> 0);
      v[pos + 1] = static_cast<uint8_t>(block_len >> 8);
      v[pos + 2] = static_cast<uint8_t>(block_len >> 16);
      pos += 3 + compression_blocksize_;
      v.insert(v.begin() + pos, 3, 0);
      uncomp_len -= compression_blocksize_;
    }
    block_len  = uncomp_len * 2 + 1;
    v[pos + 0] = static_cast<uint8_t>(block_len >> 0);
    v[pos + 1] = static_cast<uint8_t>(block_len >> 8);
    v[pos + 2] = static_cast<uint8_t>(block_len >> 16);
  }
}

writer::impl::impl(std::unique_ptr<data_sink> sink,
                   orc_writer_options const& options,
                   SingleWriteMode mode,
                   rmm::cuda_stream_view stream,
                   rmm::mr::device_memory_resource* mr)
  : _mr(mr),
    stream(stream),
    max_stripe_size{options.get_stripe_size_bytes(), options.get_stripe_size_rows()},
    row_index_stride{options.get_row_index_stride()},
    compression_kind_(to_orc_compression(options.get_compression())),
    compression_blocksize_(compression_block_size(compression_kind_)),
    stats_freq_(options.get_statistics_freq()),
    single_write_mode(mode == SingleWriteMode::YES),
    kv_meta(options.get_key_value_metadata()),
    out_sink_(std::move(sink))
{
  if (options.get_metadata()) {
    table_meta = std::make_unique<table_input_metadata>(*options.get_metadata());
  }
  init_state();
}

writer::impl::impl(std::unique_ptr<data_sink> sink,
                   chunked_orc_writer_options const& options,
                   SingleWriteMode mode,
                   rmm::cuda_stream_view stream,
                   rmm::mr::device_memory_resource* mr)
  : _mr(mr),
    stream(stream),
    max_stripe_size{options.get_stripe_size_bytes(), options.get_stripe_size_rows()},
    row_index_stride{options.get_row_index_stride()},
    compression_kind_(to_orc_compression(options.get_compression())),
    compression_blocksize_(compression_block_size(compression_kind_)),
    stats_freq_(options.get_statistics_freq()),
    single_write_mode(mode == SingleWriteMode::YES),
    kv_meta(options.get_key_value_metadata()),
    out_sink_(std::move(sink))
{
  if (options.get_metadata()) {
    table_meta = std::make_unique<table_input_metadata>(*options.get_metadata());
  }
  init_state();
}

writer::impl::~impl() { close(); }

void writer::impl::init_state()
{
  // Write file header
  out_sink_->host_write(MAGIC, std::strlen(MAGIC));
}

void pushdown_lists_null_mask(orc_column_view const& col,
                              device_span<orc_column_device_view> d_columns,
                              bitmask_type const* parent_pd_mask,
                              device_span<bitmask_type> out_mask,
                              rmm::cuda_stream_view stream)
{
  // Set all bits - correct unless there's a mismatch between offsets and null mask
  CUDF_CUDA_TRY(hipMemsetAsync(static_cast<void*>(out_mask.data()),
                                255,
                                out_mask.size() * sizeof(bitmask_type),
                                stream.value()));

  // Reset bits where a null list element has rows in the child column
  thrust::for_each_n(
    rmm::exec_policy(stream),
    thrust::make_counting_iterator(0u),
    col.size(),
    [d_columns, col_idx = col.index(), parent_pd_mask, out_mask] __device__(auto& idx) {
      auto const d_col        = d_columns[col_idx];
      auto const is_row_valid = d_col.is_valid(idx) and bit_value_or(parent_pd_mask, idx, true);
      if (not is_row_valid) {
        auto offsets                = d_col.child(lists_column_view::offsets_column_index);
        auto const child_rows_begin = offsets.element<size_type>(idx + d_col.offset());
        auto const child_rows_end   = offsets.element<size_type>(idx + 1 + d_col.offset());
        for (auto child_row = child_rows_begin; child_row < child_rows_end; ++child_row)
          clear_bit(out_mask.data(), child_row);
      }
    });
}

/**
 * @brief All pushdown masks in a table.
 *
 * Pushdown masks are applied to child column(s). Only bits of the child column null mask that
 * correspond to set pushdown mask bits are encoded into the output file. Similarly, rows where
 * pushdown mask is 0 are treated as invalid and not included in the output.
 */
struct pushdown_null_masks {
  // Owning vector for masks in device memory
  std::vector<rmm::device_uvector<bitmask_type>> data;
  // Pointers to pushdown masks in device memory. Can be same for multiple columns.
  std::vector<bitmask_type const*> masks;
};

pushdown_null_masks init_pushdown_null_masks(orc_table_view& orc_table,
                                             rmm::cuda_stream_view stream)
{
  std::vector<bitmask_type const*> mask_ptrs;
  mask_ptrs.reserve(orc_table.num_columns());
  std::vector<rmm::device_uvector<bitmask_type>> pd_masks;
  for (auto const& col : orc_table.columns) {
    // Leaf columns don't need pushdown masks
    if (col.num_children() == 0) {
      mask_ptrs.emplace_back(nullptr);
      continue;
    }
    auto const parent_pd_mask = col.is_child() ? mask_ptrs[col.parent_index()] : nullptr;
    auto const null_mask      = col.null_mask();

    if (null_mask == nullptr and parent_pd_mask == nullptr) {
      mask_ptrs.emplace_back(nullptr);
      continue;
    }
    if (col.orc_kind() == STRUCT) {
      if (null_mask != nullptr and parent_pd_mask == nullptr) {
        // Reuse own null mask
        mask_ptrs.emplace_back(null_mask);
      } else if (null_mask == nullptr and parent_pd_mask != nullptr) {
        // Reuse parent's pushdown mask
        mask_ptrs.emplace_back(parent_pd_mask);
      } else {
        // Both are nullable, allocate new pushdown mask
        pd_masks.emplace_back(num_bitmask_words(col.size()), stream);
        mask_ptrs.emplace_back(pd_masks.back().data());

        thrust::transform(rmm::exec_policy(stream),
                          null_mask,
                          null_mask + pd_masks.back().size(),
                          parent_pd_mask,
                          pd_masks.back().data(),
                          thrust::bit_and<bitmask_type>());
      }
    }
    if (col.orc_kind() == LIST or col.orc_kind() == MAP) {
      // Need a new pushdown mask unless both the parent and current column are not nullable
      auto const child_col = orc_table.column(col.child_begin()[0]);
      // pushdown mask applies to child column(s); use the child column size
      pd_masks.emplace_back(num_bitmask_words(child_col.size()), stream);
      mask_ptrs.emplace_back(pd_masks.back().data());
      pushdown_lists_null_mask(col, orc_table.d_columns, parent_pd_mask, pd_masks.back(), stream);
    }
  }

  // Attach null masks to device column views (async)
  auto const d_mask_ptrs = cudf::detail::make_device_uvector_async(mask_ptrs, stream);
  thrust::for_each_n(
    rmm::exec_policy(stream),
    thrust::make_counting_iterator(0ul),
    orc_table.num_columns(),
    [cols = device_span<orc_column_device_view>{orc_table.d_columns},
     ptrs = device_span<bitmask_type const* const>{d_mask_ptrs}] __device__(auto& idx) {
      cols[idx].pushdown_mask = ptrs[idx];
    });

  return {std::move(pd_masks), std::move(mask_ptrs)};
}

template <typename T>
struct device_stack {
  __device__ device_stack(T* stack_storage, int capacity)
    : stack(stack_storage), capacity(capacity), size(0)
  {
  }
  __device__ void push(T const& val)
  {
    cudf_assert(size < capacity and "Stack overflow");
    stack[size++] = val;
  }
  __device__ T pop()
  {
    cudf_assert(size > 0 and "Stack underflow");
    return stack[--size];
  }
  __device__ bool empty() { return size == 0; }

 private:
  T* stack;
  int capacity;
  int size;
};

orc_table_view make_orc_table_view(table_view const& table,
                                   table_device_view const& d_table,
                                   table_input_metadata const& table_meta,
                                   rmm::cuda_stream_view stream)
{
  std::vector<orc_column_view> orc_columns;
  std::vector<uint32_t> str_col_indexes;

  std::function<void(column_view const&, orc_column_view*, column_in_metadata const&)>
    append_orc_column =
      [&](column_view const& col, orc_column_view* parent_col, column_in_metadata const& col_meta) {
        int const str_idx =
          (col.type().id() == type_id::STRING) ? static_cast<int>(str_col_indexes.size()) : -1;

        auto const new_col_idx = orc_columns.size();
        orc_columns.emplace_back(new_col_idx, str_idx, parent_col, col, col_meta);
        if (orc_columns[new_col_idx].is_string()) { str_col_indexes.push_back(new_col_idx); }

        auto const kind = orc_columns[new_col_idx].orc_kind();
        if (kind == TypeKind::LIST) {
          append_orc_column(col.child(lists_column_view::child_column_index),
                            &orc_columns[new_col_idx],
                            col_meta.child(lists_column_view::child_column_index));
        } else if (kind == TypeKind::STRUCT) {
          for (auto child_idx = 0; child_idx != col.num_children(); ++child_idx) {
            append_orc_column(
              col.child(child_idx), &orc_columns[new_col_idx], col_meta.child(child_idx));
          }
        } else if (kind == TypeKind::MAP) {
          // MAP: skip to the list child - include grandchildren columns instead of children
          auto const real_parent_col   = col.child(lists_column_view::child_column_index);
          auto const& real_parent_meta = col_meta.child(lists_column_view::child_column_index);
          CUDF_EXPECTS(real_parent_meta.num_children() == 2,
                       "Map struct column should have exactly two children");
          // process MAP key
          append_orc_column(
            real_parent_col.child(0), &orc_columns[new_col_idx], real_parent_meta.child(0));
          // process MAP value
          append_orc_column(
            real_parent_col.child(1), &orc_columns[new_col_idx], real_parent_meta.child(1));
        }
      };

  for (auto col_idx = 0; col_idx < table.num_columns(); ++col_idx) {
    append_orc_column(table.column(col_idx), nullptr, table_meta.column_metadata[col_idx]);
  }

  std::vector<TypeKind> type_kinds;
  type_kinds.reserve(orc_columns.size());
  std::transform(
    orc_columns.cbegin(), orc_columns.cend(), std::back_inserter(type_kinds), [](auto& orc_column) {
      return orc_column.orc_kind();
    });
  auto const d_type_kinds = cudf::detail::make_device_uvector_async(type_kinds, stream);

  rmm::device_uvector<orc_column_device_view> d_orc_columns(orc_columns.size(), stream);
  using stack_value_type = thrust::pair<column_device_view const*, thrust::optional<uint32_t>>;
  rmm::device_uvector<stack_value_type> stack_storage(orc_columns.size(), stream);

  // pre-order append ORC device columns
  cudf::detail::device_single_thread(
    [d_orc_cols         = device_span<orc_column_device_view>{d_orc_columns},
     d_type_kinds       = device_span<TypeKind const>{d_type_kinds},
     d_table            = d_table,
     stack_storage      = stack_storage.data(),
     stack_storage_size = stack_storage.size()] __device__() {
      device_stack stack(stack_storage, stack_storage_size);

      thrust::for_each(thrust::seq,
                       thrust::make_reverse_iterator(d_table.end()),
                       thrust::make_reverse_iterator(d_table.begin()),
                       [&stack](column_device_view const& c) {
                         stack.push({&c, thrust::nullopt});
                       });

      uint32_t idx = 0;
      while (not stack.empty()) {
        auto [col, parent] = stack.pop();
        d_orc_cols[idx]    = orc_column_device_view{*col, parent};

        if (d_type_kinds[idx] == TypeKind::MAP) {
          // Skip to the list child - do not include the child column, just grandchildren columns
          col = &col->children()[lists_column_view::child_column_index];
        }

        if (col->type().id() == type_id::LIST) {
          stack.push({&col->children()[lists_column_view::child_column_index], idx});
        } else if (col->type().id() == type_id::STRUCT) {
          thrust::for_each(thrust::seq,
                           thrust::make_reverse_iterator(col->children().end()),
                           thrust::make_reverse_iterator(col->children().begin()),
                           [&stack, idx](column_device_view const& c) {
                             stack.push({&c, idx});
                           });
        }
        ++idx;
      }
    },
    stream);

  return {std::move(orc_columns),
          std::move(d_orc_columns),
          str_col_indexes,
          cudf::detail::make_device_uvector_sync(str_col_indexes, stream)};
}

hostdevice_2dvector<rowgroup_rows> calculate_rowgroup_bounds(orc_table_view const& orc_table,
                                                             size_type rowgroup_size,
                                                             rmm::cuda_stream_view stream)
{
  auto const num_rowgroups =
    cudf::util::div_rounding_up_unsafe<size_t, size_t>(orc_table.num_rows(), rowgroup_size);

  hostdevice_2dvector<rowgroup_rows> rowgroup_bounds(
    num_rowgroups, orc_table.num_columns(), stream);
  thrust::for_each_n(
    rmm::exec_policy(stream),
    thrust::make_counting_iterator(0ul),
    num_rowgroups,
    [cols      = device_span<orc_column_device_view const>{orc_table.d_columns},
     rg_bounds = device_2dspan<rowgroup_rows>{rowgroup_bounds},
     rowgroup_size] __device__(auto rg_idx) mutable {
      thrust::transform(
        thrust::seq, cols.begin(), cols.end(), rg_bounds[rg_idx].begin(), [&](auto const& col) {
          // Root column
          if (!col.parent_index.has_value()) {
            size_type const rows_begin = rg_idx * rowgroup_size;
            auto const rows_end = thrust::min<size_type>((rg_idx + 1) * rowgroup_size, col.size());
            return rowgroup_rows{rows_begin, rows_end};
          } else {
            // Child column
            auto const parent_index           = *col.parent_index;
            orc_column_device_view parent_col = cols[parent_index];
            auto const parent_rg              = rg_bounds[rg_idx][parent_index];
            if (parent_col.type().id() != type_id::LIST) {
              auto const offset_diff = parent_col.offset() - col.offset();
              return rowgroup_rows{parent_rg.begin + offset_diff, parent_rg.end + offset_diff};
            }

            auto offsets = parent_col.child(lists_column_view::offsets_column_index);
            auto const rows_begin =
              offsets.element<size_type>(parent_rg.begin + parent_col.offset()) - col.offset();
            auto const rows_end =
              offsets.element<size_type>(parent_rg.end + parent_col.offset()) - col.offset();

            return rowgroup_rows{rows_begin, rows_end};
          }
        });
    });
  rowgroup_bounds.device_to_host(stream, true);

  return rowgroup_bounds;
}

// returns host vector of per-rowgroup sizes
encoder_decimal_info decimal_chunk_sizes(orc_table_view& orc_table,
                                         file_segmentation const& segmentation,
                                         rmm::cuda_stream_view stream)
{
  std::map<uint32_t, rmm::device_uvector<uint32_t>> elem_sizes;
  // Compute per-element offsets (within each row group) on the device
  for (auto& orc_col : orc_table.columns) {
    if (orc_col.orc_kind() == DECIMAL) {
      auto& current_sizes =
        elem_sizes.insert({orc_col.index(), rmm::device_uvector<uint32_t>(orc_col.size(), stream)})
          .first->second;
      thrust::tabulate(rmm::exec_policy(stream),
                       current_sizes.begin(),
                       current_sizes.end(),
                       [d_cols  = device_span<orc_column_device_view const>{orc_table.d_columns},
                        col_idx = orc_col.index()] __device__(auto idx) {
                         auto const& col          = d_cols[col_idx];
                         auto const pushdown_mask = [&]() -> cudf::bitmask_type const* {
                           auto const parent_index = d_cols[col_idx].parent_index;
                           if (!parent_index.has_value()) return nullptr;
                           return d_cols[parent_index.value()].pushdown_mask;
                         }();

                         if (col.is_null(idx) or not bit_value_or(pushdown_mask, idx, true))
                           return 0u;

                         __int128_t const element =
                           col.type().id() == type_id::DECIMAL32   ? col.element<int32_t>(idx)
                           : col.type().id() == type_id::DECIMAL64 ? col.element<int64_t>(idx)
                                                                   : col.element<__int128_t>(idx);

                         __int128_t const sign      = (element < 0) ? 1 : 0;
                         __uint128_t zigzaged_value = ((element ^ -sign) * 2) + sign;

                         uint32_t encoded_length = 1;
                         while (zigzaged_value > 127) {
                           zigzaged_value >>= 7u;
                           ++encoded_length;
                         }
                         return encoded_length;
                       });

      // Compute element offsets within each row group
      thrust::for_each_n(rmm::exec_policy(stream),
                         thrust::make_counting_iterator(0ul),
                         segmentation.num_rowgroups(),
                         [sizes     = device_span<uint32_t>{current_sizes},
                          rg_bounds = device_2dspan<rowgroup_rows const>{segmentation.rowgroups},
                          col_idx   = orc_col.index()] __device__(auto rg_idx) {
                           auto const& range = rg_bounds[rg_idx][col_idx];
                           thrust::inclusive_scan(thrust::seq,
                                                  sizes.begin() + range.begin,
                                                  sizes.begin() + range.end,
                                                  sizes.begin() + range.begin);
                         });

      orc_col.attach_decimal_offsets(current_sizes.data());
    }
  }
  if (elem_sizes.empty()) return {};

  // Gather the row group sizes and copy to host
  auto d_tmp_rowgroup_sizes = rmm::device_uvector<uint32_t>(segmentation.num_rowgroups(), stream);
  std::map<uint32_t, std::vector<uint32_t>> rg_sizes;
  for (auto const& [col_idx, esizes] : elem_sizes) {
    // Copy last elem in each row group - equal to row group size
    thrust::tabulate(rmm::exec_policy(stream),
                     d_tmp_rowgroup_sizes.begin(),
                     d_tmp_rowgroup_sizes.end(),
                     [src       = esizes.data(),
                      col_idx   = col_idx,
                      rg_bounds = device_2dspan<rowgroup_rows const>{
                        segmentation.rowgroups}] __device__(auto idx) {
                       return src[rg_bounds[idx][col_idx].end - 1];
                     });

    rg_sizes[col_idx] = cudf::detail::make_std_vector_async(d_tmp_rowgroup_sizes, stream);
  }

  return {std::move(elem_sizes), std::move(rg_sizes)};
}

std::map<uint32_t, size_t> decimal_column_sizes(
  std::map<uint32_t, std::vector<uint32_t>> const& chunk_sizes)
{
  std::map<uint32_t, size_t> column_sizes;
  std::transform(chunk_sizes.cbegin(),
                 chunk_sizes.cend(),
                 std::inserter(column_sizes, column_sizes.end()),
                 [](auto const& chunk_size) -> std::pair<uint32_t, size_t> {
                   return {
                     chunk_size.first,
                     std::accumulate(chunk_size.second.cbegin(), chunk_size.second.cend(), 0lu)};
                 });
  return column_sizes;
}

string_dictionaries allocate_dictionaries(orc_table_view const& orc_table,
                                          host_2dspan<rowgroup_rows const> rowgroup_bounds,
                                          rmm::cuda_stream_view stream)
{
  thrust::host_vector<bool> is_dict_enabled(orc_table.num_columns());
  for (auto col_idx : orc_table.string_column_indices)
    is_dict_enabled[col_idx] = std::all_of(
      thrust::make_counting_iterator(0ul),
      thrust::make_counting_iterator(rowgroup_bounds.size().first),
      [&](auto rg_idx) {
        return rowgroup_bounds[rg_idx][col_idx].size() < std::numeric_limits<uint16_t>::max();
      });

  std::vector<rmm::device_uvector<uint32_t>> data;
  std::transform(orc_table.string_column_indices.begin(),
                 orc_table.string_column_indices.end(),
                 std::back_inserter(data),
                 [&](auto& idx) {
                   return cudf::detail::make_zeroed_device_uvector_async<uint32_t>(
                     orc_table.columns[idx].size(), stream);
                 });
  std::vector<rmm::device_uvector<uint32_t>> index;
  std::transform(orc_table.string_column_indices.begin(),
                 orc_table.string_column_indices.end(),
                 std::back_inserter(index),
                 [&](auto& idx) {
                   return cudf::detail::make_zeroed_device_uvector_async<uint32_t>(
                     orc_table.columns[idx].size(), stream);
                 });
  stream.synchronize();

  std::vector<device_span<uint32_t>> data_ptrs;
  std::transform(data.begin(), data.end(), std::back_inserter(data_ptrs), [](auto& uvec) {
    return device_span<uint32_t>{uvec};
  });
  std::vector<device_span<uint32_t>> index_ptrs;
  std::transform(index.begin(), index.end(), std::back_inserter(index_ptrs), [](auto& uvec) {
    return device_span<uint32_t>{uvec};
  });

  return {std::move(data),
          std::move(index),
          cudf::detail::make_device_uvector_sync(data_ptrs, stream),
          cudf::detail::make_device_uvector_sync(index_ptrs, stream),
          std::move(is_dict_enabled)};
}

struct string_length_functor {
  __device__ inline size_type operator()(int const i) const
  {
    // we translate from 0 -> num_chunks * 2 because each statistic has a min and max
    // string and we need to calculate lengths for both.
    if (i >= num_chunks * 2) return 0;

    // min strings are even values, max strings are odd values of i
    auto const should_copy_min = i % 2 == 0;
    // index of the chunk
    auto const idx = i / 2;
    auto& str_val  = should_copy_min ? stripe_stat_chunks[idx].min_value.str_val
                                     : stripe_stat_chunks[idx].max_value.str_val;
    auto const str = stripe_stat_merge[idx].stats_dtype == dtype_string;
    return str ? str_val.length : 0;
  }

  int const num_chunks;
  statistics_chunk const* stripe_stat_chunks;
  statistics_merge_group const* stripe_stat_merge;
};

__global__ void copy_string_data(char* string_pool,
                                 size_type* offsets,
                                 statistics_chunk* chunks,
                                 statistics_merge_group const* groups)
{
  auto const idx = blockIdx.x / 2;
  if (groups[idx].stats_dtype == dtype_string) {
    // min strings are even values, max strings are odd values of i
    auto const should_copy_min = blockIdx.x % 2 == 0;
    auto& str_val = should_copy_min ? chunks[idx].min_value.str_val : chunks[idx].max_value.str_val;
    auto dst      = &string_pool[offsets[blockIdx.x]];
    auto src      = str_val.ptr;

    for (int i = threadIdx.x; i < str_val.length; i += blockDim.x) {
      dst[i] = src[i];
    }
    if (threadIdx.x == 0) { str_val.ptr = dst; }
  }
}

size_t max_compression_output_size(CompressionKind compression_kind, uint32_t compression_blocksize)
{
  if (compression_kind == NONE) return 0;

  return compress_max_output_chunk_size(to_nvcomp_compression_type(compression_kind),
                                        compression_blocksize);
}

void writer::impl::persisted_statistics::persist(int num_table_rows,
                                                 bool single_write_mode,
                                                 intermediate_statistics& intermediate_stats,
                                                 rmm::cuda_stream_view stream)
{
  if (not single_write_mode) {
    // persist the strings in the chunks into a string pool and update pointers
    auto const num_chunks = static_cast<int>(intermediate_stats.stripe_stat_chunks.size());
    // min offset and max offset + 1 for total size
    rmm::device_uvector<size_type> offsets((num_chunks * 2) + 1, stream);

    auto iter = cudf::detail::make_counting_transform_iterator(
      0,
      string_length_functor{num_chunks,
                            intermediate_stats.stripe_stat_chunks.data(),
                            intermediate_stats.stripe_stat_merge.device_ptr()});
    thrust::exclusive_scan(rmm::exec_policy(stream), iter, iter + offsets.size(), offsets.begin());

    // pull size back to host
    auto const total_string_pool_size = offsets.element(num_chunks * 2, stream);
    if (total_string_pool_size > 0) {
      rmm::device_uvector<char> string_pool(total_string_pool_size, stream);

      // offsets describes where in the string pool each string goes. Going with the simple
      // approach for now, but it is possible something fancier with breaking up each thread into
      // copying x bytes instead of a single string is the better method since we are dealing in
      // min/max strings they almost certainly will not be uniform length.
      copy_string_data<<<num_chunks * 2, 256, 0, stream.value()>>>(
        string_pool.data(),
        offsets.data(),
        intermediate_stats.stripe_stat_chunks.data(),
        intermediate_stats.stripe_stat_merge.device_ptr());
      string_pools.emplace_back(std::move(string_pool));
    }
  }

  stripe_stat_chunks.emplace_back(std::move(intermediate_stats.stripe_stat_chunks));
  stripe_stat_merge.emplace_back(std::move(intermediate_stats.stripe_stat_merge));
  stats_dtypes = std::move(intermediate_stats.stats_dtypes);
  col_types    = std::move(intermediate_stats.col_types);
  num_rows     = num_table_rows;
}

void writer::impl::write(table_view const& table)
{
  CUDF_EXPECTS(not closed, "Data has already been flushed to out and closed");
  auto const num_rows = table.num_rows();

  if (not table_meta) { table_meta = std::make_unique<table_input_metadata>(table); }

  // Fill unnamed columns' names in table_meta
  std::function<void(column_in_metadata&, std::string)> add_default_name =
    [&](column_in_metadata& col_meta, std::string default_name) {
      if (col_meta.get_name().empty()) col_meta.set_name(default_name);
      for (size_type i = 0; i < col_meta.num_children(); ++i) {
        add_default_name(col_meta.child(i), std::to_string(i));
      }
    };
  for (size_t i = 0; i < table_meta->column_metadata.size(); ++i) {
    add_default_name(table_meta->column_metadata[i], "_col" + std::to_string(i));
  }

  auto const d_table = table_device_view::create(table, stream);

  auto orc_table = make_orc_table_view(table, *d_table, *table_meta, stream);

  auto const pd_masks = init_pushdown_null_masks(orc_table, stream);

  auto rowgroup_bounds = calculate_rowgroup_bounds(orc_table, row_index_stride, stream);

  // Build per-column dictionary indices
  auto dictionaries = allocate_dictionaries(orc_table, rowgroup_bounds, stream);
  hostdevice_2dvector<gpu::DictionaryChunk> dict(
    rowgroup_bounds.size().first, orc_table.num_string_columns(), stream);
  if (not dict.is_empty()) {
    init_dictionaries(orc_table,
                      rowgroup_bounds,
                      dictionaries.d_data_view,
                      dictionaries.d_index_view,
                      &dict,
                      stream);
  }

  // Decide stripe boundaries based on rowgroups and dict chunks
  auto const segmentation =
    calculate_segmentation(orc_table.columns, std::move(rowgroup_bounds), max_stripe_size);

  // Build stripe-level dictionaries
  hostdevice_2dvector<gpu::StripeDictionary> stripe_dict(
    segmentation.num_stripes(), orc_table.num_string_columns(), stream);
  if (not stripe_dict.is_empty()) {
    build_dictionaries(orc_table,
                       segmentation.stripes,
                       dict,
                       dictionaries.index,
                       dictionaries.dictionary_enabled,
                       stripe_dict);
  }

  auto dec_chunk_sizes = decimal_chunk_sizes(orc_table, segmentation, stream);

  auto const uncompressed_block_align = uncomp_block_alignment(compression_kind_);
  auto const compressed_block_align   = comp_block_alignment(compression_kind_);
  auto streams =
    create_streams(orc_table.columns, segmentation, decimal_column_sizes(dec_chunk_sizes.rg_sizes));
  auto enc_data = encode_columns(orc_table,
                                 std::move(dictionaries),
                                 std::move(dec_chunk_sizes),
                                 segmentation,
                                 streams,
                                 uncompressed_block_align,
                                 stream);

  // Assemble individual disparate column chunks into contiguous data streams
  size_type const num_index_streams = (orc_table.num_columns() + 1);
  const auto num_data_streams       = streams.size() - num_index_streams;
  hostdevice_2dvector<gpu::StripeStream> strm_descs(
    segmentation.num_stripes(), num_data_streams, stream);
  auto stripes = gather_stripes(num_index_streams, segmentation, &enc_data.streams, &strm_descs);

  if (num_rows > 0) {
    // Allocate intermediate output stream buffer
    size_t compressed_bfr_size   = 0;
    size_t num_compressed_blocks = 0;

    auto const max_compressed_block_size =
      max_compression_output_size(compression_kind_, compression_blocksize_);
    auto const padded_max_compressed_block_size =
      util::round_up_unsafe<size_t>(max_compressed_block_size, compressed_block_align);
    auto const padded_block_header_size =
      util::round_up_unsafe<size_t>(block_header_size, compressed_block_align);

    auto stream_output = [&]() {
      size_t max_stream_size = 0;
      bool all_device_write  = true;

      for (auto& ss : strm_descs.host_view().flat_view()) {
        if (!out_sink_->is_device_write_preferred(ss.stream_size)) { all_device_write = false; }
        size_t stream_size = ss.stream_size;
        if (compression_kind_ != NONE) {
          ss.first_block = num_compressed_blocks;
          ss.bfr_offset  = compressed_bfr_size;

          auto num_blocks = std::max<uint32_t>(
            (stream_size + compression_blocksize_ - 1) / compression_blocksize_, 1);
          stream_size += num_blocks * block_header_size;
          num_compressed_blocks += num_blocks;
          compressed_bfr_size +=
            (padded_block_header_size + padded_max_compressed_block_size) * num_blocks;
        }
        max_stream_size = std::max(max_stream_size, stream_size);
      }

      if (all_device_write) {
        return pinned_buffer<uint8_t>{nullptr, hipHostFree};
      } else {
        return pinned_buffer<uint8_t>{[](size_t size) {
                                        uint8_t* ptr = nullptr;
                                        CUDF_CUDA_TRY(hipHostMalloc(&ptr, size));
                                        return ptr;
                                      }(max_stream_size),
                                      hipHostFree};
      }
    }();

    // Compress the data streams
    rmm::device_buffer compressed_data(compressed_bfr_size, stream);
    hostdevice_vector<compression_result> comp_results(num_compressed_blocks, stream);
    thrust::fill(rmm::exec_policy(stream),
                 comp_results.d_begin(),
                 comp_results.d_end(),
                 compression_result{0, compression_status::FAILURE});
    if (compression_kind_ != NONE) {
      strm_descs.host_to_device(stream);
      gpu::CompressOrcDataStreams(static_cast<uint8_t*>(compressed_data.data()),
                                  num_compressed_blocks,
                                  compression_kind_,
                                  compression_blocksize_,
                                  max_compressed_block_size,
                                  compressed_block_align,
                                  strm_descs,
                                  enc_data.streams,
                                  comp_results,
                                  stream);
      strm_descs.device_to_host(stream);
      comp_results.device_to_host(stream, true);
    }

    ProtobufWriter pbw_(&buffer_);

    auto intermediate_stats = gather_statistic_blobs(stats_freq_, orc_table, segmentation);

    if (intermediate_stats.stripe_stat_chunks.size() > 0) {
      persisted_stripe_statistics.persist(
        orc_table.num_rows(), single_write_mode, intermediate_stats, stream);
    }

    // Write stripes
    std::vector<std::future<void>> write_tasks;
    for (size_t stripe_id = 0; stripe_id < stripes.size(); ++stripe_id) {
      auto& stripe = stripes[stripe_id];

      stripe.offset = out_sink_->bytes_written();

      // Column (skippable) index streams appear at the start of the stripe
      for (size_type stream_id = 0; stream_id < num_index_streams; ++stream_id) {
        write_index_stream(stripe_id,
                           stream_id,
                           orc_table.columns,
                           segmentation,
                           enc_data.streams,
                           strm_descs,
                           comp_results,
                           intermediate_stats.rowgroup_blobs,
                           &stripe,
                           &streams,
                           &pbw_);
      }

      // Column data consisting one or more separate streams
      for (auto const& strm_desc : strm_descs[stripe_id]) {
        write_tasks.push_back(write_data_stream(
          strm_desc,
          enc_data.streams[strm_desc.column_id][segmentation.stripes[stripe_id].first],
          static_cast<uint8_t const*>(compressed_data.data()),
          stream_output.get(),
          &stripe,
          &streams));
      }

      // Write stripefooter consisting of stream information
      StripeFooter sf;
      sf.streams = streams;
      sf.columns.resize(orc_table.num_columns() + 1);
      sf.columns[0].kind = DIRECT;
      for (size_t i = 1; i < sf.columns.size(); ++i) {
        sf.columns[i].kind = orc_table.column(i - 1).orc_encoding();
        sf.columns[i].dictionarySize =
          (sf.columns[i].kind == DICTIONARY_V2)
            ? orc_table.column(i - 1).host_stripe_dict(stripe_id)->num_strings
            : 0;
        if (orc_table.column(i - 1).orc_kind() == TIMESTAMP) { sf.writerTimezone = "UTC"; }
      }
      buffer_.resize((compression_kind_ != NONE) ? 3 : 0);
      pbw_.write(sf);
      stripe.footerLength = buffer_.size();
      if (compression_kind_ != NONE) {
        uint32_t uncomp_sf_len = (stripe.footerLength - 3) * 2 + 1;
        buffer_[0]             = static_cast<uint8_t>(uncomp_sf_len >> 0);
        buffer_[1]             = static_cast<uint8_t>(uncomp_sf_len >> 8);
        buffer_[2]             = static_cast<uint8_t>(uncomp_sf_len >> 16);
      }
      out_sink_->host_write(buffer_.data(), buffer_.size());
    }
    for (auto const& task : write_tasks) {
      task.wait();
    }
  }
  if (ff.headerLength == 0) {
    // First call
    ff.headerLength   = std::strlen(MAGIC);
    ff.rowIndexStride = row_index_stride;
    ff.types.resize(1 + orc_table.num_columns());
    ff.types[0].kind = STRUCT;
    for (auto const& column : orc_table.columns) {
      if (!column.is_child()) {
        ff.types[0].subtypes.emplace_back(column.id());
        ff.types[0].fieldNames.emplace_back(column.orc_name());
      }
    }
    for (auto const& column : orc_table.columns) {
      auto& schema_type = ff.types[column.id()];
      schema_type.kind  = column.orc_kind();
      if (column.orc_kind() == DECIMAL) {
        schema_type.scale     = static_cast<uint32_t>(column.scale());
        schema_type.precision = column.precision();
      }
      std::transform(column.child_begin(),
                     column.child_end(),
                     std::back_inserter(schema_type.subtypes),
                     [&](auto const& child_idx) { return orc_table.column(child_idx).id(); });
      if (column.orc_kind() == STRUCT) {
        std::transform(column.child_begin(),
                       column.child_end(),
                       std::back_inserter(schema_type.fieldNames),
                       [&](auto const& child_idx) {
                         return std::string{orc_table.column(child_idx).orc_name()};
                       });
      }
    }
  } else {
    // verify the user isn't passing mismatched tables
    CUDF_EXPECTS(ff.types.size() == 1 + orc_table.num_columns(),
                 "Mismatch in table structure between multiple calls to write");
    CUDF_EXPECTS(
      std::all_of(orc_table.columns.cbegin(),
                  orc_table.columns.cend(),
                  [&](auto const& col) { return ff.types[col.id()].kind == col.orc_kind(); }),
      "Mismatch in column types between multiple calls to write");
  }
  ff.stripes.insert(ff.stripes.end(),
                    std::make_move_iterator(stripes.begin()),
                    std::make_move_iterator(stripes.end()));
  ff.numberOfRows += num_rows;
}

void writer::impl::close()
{
  if (closed) { return; }
  closed = true;
  ProtobufWriter pbw_(&buffer_);
  PostScript ps;

  auto const statistics = finish_statistic_blobs(ff.stripes.size(), persisted_stripe_statistics);

  // File-level statistics
  if (not statistics.file_level.empty()) {
    buffer_.resize(0);
    pbw_.put_uint(encode_field_number<size_type>(1));
    pbw_.put_uint(persisted_stripe_statistics.num_rows);
    // First entry contains total number of rows
    ff.statistics.reserve(ff.types.size());
    ff.statistics.emplace_back(std::move(buffer_));
    // Add file stats, stored after stripe stats in `column_stats`
    ff.statistics.insert(ff.statistics.end(),
                         std::make_move_iterator(statistics.file_level.begin()),
                         std::make_move_iterator(statistics.file_level.end()));
  }

  // Stripe-level statistics
  if (not statistics.stripe_level.empty()) {
    md.stripeStats.resize(ff.stripes.size());
    for (size_t stripe_id = 0; stripe_id < ff.stripes.size(); stripe_id++) {
      md.stripeStats[stripe_id].colStats.resize(ff.types.size());
      buffer_.resize(0);
      pbw_.put_uint(encode_field_number<size_type>(1));
      pbw_.put_uint(ff.stripes[stripe_id].numberOfRows);
      md.stripeStats[stripe_id].colStats[0] = std::move(buffer_);
      for (size_t col_idx = 0; col_idx < ff.types.size() - 1; col_idx++) {
        size_t idx                                      = ff.stripes.size() * col_idx + stripe_id;
        md.stripeStats[stripe_id].colStats[1 + col_idx] = std::move(statistics.stripe_level[idx]);
      }
    }
  }

  persisted_stripe_statistics.clear();

  ff.contentLength = out_sink_->bytes_written();
  std::transform(
    kv_meta.begin(), kv_meta.end(), std::back_inserter(ff.metadata), [&](auto const& udata) {
      return UserMetadataItem{udata.first, udata.second};
    });

  // Write statistics metadata
  if (md.stripeStats.size() != 0) {
    buffer_.resize((compression_kind_ != NONE) ? 3 : 0);
    pbw_.write(md);
    add_uncompressed_block_headers(buffer_);
    ps.metadataLength = buffer_.size();
    out_sink_->host_write(buffer_.data(), buffer_.size());
  } else {
    ps.metadataLength = 0;
  }
  buffer_.resize((compression_kind_ != NONE) ? 3 : 0);
  pbw_.write(ff);
  add_uncompressed_block_headers(buffer_);

  // Write postscript metadata
  ps.footerLength         = buffer_.size();
  ps.compression          = compression_kind_;
  ps.compressionBlockSize = compression_blocksize_;
  ps.version              = {0, 12};
  ps.magic                = MAGIC;
  const auto ps_length    = static_cast<uint8_t>(pbw_.write(ps));
  buffer_.push_back(ps_length);
  out_sink_->host_write(buffer_.data(), buffer_.size());
  out_sink_->flush();
}

// Forward to implementation
writer::writer(std::unique_ptr<data_sink> sink,
               orc_writer_options const& options,
               SingleWriteMode mode,
               rmm::cuda_stream_view stream,
               rmm::mr::device_memory_resource* mr)
  : _impl(std::make_unique<impl>(std::move(sink), options, mode, stream, mr))
{
}

// Forward to implementation
writer::writer(std::unique_ptr<data_sink> sink,
               chunked_orc_writer_options const& options,
               SingleWriteMode mode,
               rmm::cuda_stream_view stream,
               rmm::mr::device_memory_resource* mr)
  : _impl(std::make_unique<impl>(std::move(sink), options, mode, stream, mr))
{
}

// Destructor within this translation unit
writer::~writer() = default;

// Forward to implementation
void writer::write(table_view const& table) { _impl->write(table); }

// Forward to implementation
void writer::close() { _impl->close(); }

}  // namespace orc
}  // namespace detail
}  // namespace io
}  // namespace cudf
