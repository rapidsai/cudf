#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019-2024, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

/**
 * @file writer_impl.cu
 * @brief cuDF-IO ORC writer class implementation
 */

#include "io/comp/nvcomp_adapter.hpp"
#include "io/statistics/column_statistics.cuh"
#include "io/utilities/column_utils.cuh"
#include "writer_impl.hpp"

#include <cudf/detail/iterator.cuh>
#include <cudf/detail/null_mask.hpp>
#include <cudf/detail/utilities/cuda.cuh>
#include <cudf/detail/utilities/stream_pool.hpp>
#include <cudf/detail/utilities/vector_factories.hpp>
#include <cudf/strings/strings_column_view.hpp>
#include <cudf/utilities/bit.hpp>
#include <cudf/utilities/span.hpp>

#include <rmm/cuda_stream_view.hpp>
#include <rmm/device_buffer.hpp>
#include <rmm/device_uvector.hpp>

#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/memcpy_async.h>
#include <cuda/std/climits>
#include <cuda/std/limits>
#include <thrust/execution_policy.h>
#include <thrust/extrema.h>
#include <thrust/for_each.h>
#include <thrust/functional.h>
#include <thrust/host_vector.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/iterator/reverse_iterator.h>
#include <thrust/iterator/transform_iterator.h>
#include <thrust/optional.h>
#include <thrust/pair.h>
#include <thrust/reduce.h>
#include <thrust/scan.h>
#include <thrust/sequence.h>
#include <thrust/sort.h>
#include <thrust/tabulate.h>
#include <thrust/transform.h>

#include <algorithm>
#include <cstring>
#include <numeric>
#include <tuple>
#include <utility>

namespace cudf::io::orc::detail {

template <typename T>
[[nodiscard]] constexpr int varint_size(T val)
{
  auto len = 1u;
  while (val > 0x7f) {
    val >>= 7;
    ++len;
  }
  return len;
}

struct row_group_index_info {
  int32_t pos       = -1;  // Position
  int32_t blk_pos   = -1;  // Block Position
  int32_t comp_pos  = -1;  // Compressed Position
  int32_t comp_size = -1;  // Compressed size
};

namespace {

/**
 * @brief Translates ORC compression to nvCOMP compression
 */
auto to_nvcomp_compression_type(CompressionKind compression_kind)
{
  if (compression_kind == SNAPPY) return nvcomp::compression_type::SNAPPY;
  if (compression_kind == ZLIB) return nvcomp::compression_type::DEFLATE;
  if (compression_kind == ZSTD) return nvcomp::compression_type::ZSTD;
  if (compression_kind == LZ4) return nvcomp::compression_type::LZ4;
  CUDF_FAIL("Unsupported compression type");
}

/**
 * @brief Translates cuDF compression to ORC compression
 */
orc::CompressionKind to_orc_compression(compression_type compression)
{
  switch (compression) {
    case compression_type::AUTO:
    case compression_type::SNAPPY: return orc::CompressionKind::SNAPPY;
    case compression_type::ZLIB: return orc::CompressionKind::ZLIB;
    case compression_type::ZSTD: return orc::CompressionKind::ZSTD;
    case compression_type::LZ4: return orc::CompressionKind::LZ4;
    case compression_type::NONE: return orc::CompressionKind::NONE;
    default: CUDF_FAIL("Unsupported compression type");
  }
}

/**
 * @brief Returns the block size for a given compression kind.
 */
constexpr size_t compression_block_size(orc::CompressionKind compression)
{
  if (compression == orc::CompressionKind::NONE) { return 0; }

  auto const ncomp_type   = to_nvcomp_compression_type(compression);
  auto const nvcomp_limit = nvcomp::is_compression_disabled(ncomp_type)
                              ? std::nullopt
                              : nvcomp::compress_max_allowed_chunk_size(ncomp_type);

  constexpr size_t max_block_size = 256 * 1024;
  return std::min(nvcomp_limit.value_or(max_block_size), max_block_size);
}

/**
 * @brief Translates cuDF dtype to ORC datatype
 */
constexpr orc::TypeKind to_orc_type(cudf::type_id id, bool list_column_as_map)
{
  switch (id) {
    case cudf::type_id::INT8: return TypeKind::BYTE;
    case cudf::type_id::INT16: return TypeKind::SHORT;
    case cudf::type_id::INT32: return TypeKind::INT;
    case cudf::type_id::INT64: return TypeKind::LONG;
    case cudf::type_id::FLOAT32: return TypeKind::FLOAT;
    case cudf::type_id::FLOAT64: return TypeKind::DOUBLE;
    case cudf::type_id::BOOL8: return TypeKind::BOOLEAN;
    case cudf::type_id::TIMESTAMP_DAYS: return TypeKind::DATE;
    case cudf::type_id::TIMESTAMP_SECONDS:
    case cudf::type_id::TIMESTAMP_MICROSECONDS:
    case cudf::type_id::TIMESTAMP_MILLISECONDS:
    case cudf::type_id::TIMESTAMP_NANOSECONDS: return TypeKind::TIMESTAMP;
    case cudf::type_id::STRING: return TypeKind::STRING;
    case cudf::type_id::DECIMAL32:
    case cudf::type_id::DECIMAL64:
    case cudf::type_id::DECIMAL128: return TypeKind::DECIMAL;
    case cudf::type_id::LIST: return list_column_as_map ? TypeKind::MAP : TypeKind::LIST;
    case cudf::type_id::STRUCT: return TypeKind::STRUCT;
    default: return TypeKind::INVALID_TYPE_KIND;
  }
}

/**
 * @brief Translates time unit to nanoscale multiple.
 */
constexpr int32_t to_clockscale(cudf::type_id timestamp_id)
{
  switch (timestamp_id) {
    case cudf::type_id::TIMESTAMP_SECONDS: return 9;
    case cudf::type_id::TIMESTAMP_MILLISECONDS: return 6;
    case cudf::type_id::TIMESTAMP_MICROSECONDS: return 3;
    case cudf::type_id::TIMESTAMP_NANOSECONDS:
    default: return 0;
  }
}

/**
 * @brief Returns the precision of the given decimal type.
 */
constexpr auto orc_precision(cudf::type_id decimal_id)
{
  using namespace numeric;
  switch (decimal_id) {
    case cudf::type_id::DECIMAL32: return cuda::std::numeric_limits<decimal32::rep>::digits10;
    case cudf::type_id::DECIMAL64: return cuda::std::numeric_limits<decimal64::rep>::digits10;
    case cudf::type_id::DECIMAL128: return cuda::std::numeric_limits<decimal128::rep>::digits10;
    default: return 0;
  }
}

}  // namespace

/**
 * @brief Helper class that adds ORC-specific column info
 */
class orc_column_view {
 public:
  /**
   * @brief Constructor that extracts out the string position + length pairs
   * for building dictionaries for string columns
   */
  explicit orc_column_view(uint32_t index,
                           int str_idx,
                           orc_column_view* parent,
                           column_view const& col,
                           column_in_metadata const& metadata)
    : cudf_column{col},
      _index{index},
      _str_idx{str_idx},
      _is_child{parent != nullptr},
      _type_width{cudf::is_fixed_width(col.type()) ? cudf::size_of(col.type()) : 0},
      _type_kind{to_orc_type(col.type().id(), metadata.is_map())},
      _scale{(_type_kind == TypeKind::DECIMAL) ? -col.type().scale()
                                               : to_clockscale(col.type().id())},
      _precision{metadata.is_decimal_precision_set() ? metadata.get_decimal_precision()
                                                     : orc_precision(col.type().id())},
      name{metadata.get_name()}
  {
    if (metadata.is_nullability_defined()) { nullable_from_metadata = metadata.nullable(); }
    if (parent != nullptr) {
      parent->add_child(_index);
      _parent_index = parent->index();
    }

    if (_type_kind == TypeKind::MAP) {
      auto const struct_col = col.child(lists_column_view::child_column_index);
      CUDF_EXPECTS(struct_col.null_count() == 0,
                   "struct column of a MAP column should not have null elements");
      CUDF_EXPECTS(struct_col.num_children() == 2, "MAP column must have two child columns");
    }
  }

  void add_child(uint32_t child_idx) { children.emplace_back(child_idx); }

  auto type() const noexcept { return cudf_column.type(); }
  auto is_string() const noexcept { return cudf_column.type().id() == type_id::STRING; }

  void attach_rowgroup_char_counts(host_span<size_type const> counts)
  {
    rowgroup_char_counts = counts;
  }

  [[nodiscard]] auto rowgroup_char_count(size_type rg_idx) const
  {
    return rowgroup_char_counts[rg_idx];
  }

  [[nodiscard]] auto char_count() const
  {
    return std::accumulate(rowgroup_char_counts.begin(), rowgroup_char_counts.end(), size_type{0});
  }

  [[nodiscard]] auto const& decimal_offsets() const { return d_decimal_offsets; }
  void attach_decimal_offsets(uint32_t* sizes_ptr) { d_decimal_offsets = sizes_ptr; }

  void attach_stripe_dicts(host_span<gpu::stripe_dictionary const> host_stripe_dicts,
                           device_span<gpu::stripe_dictionary const> dev_stripe_dicts)
  {
    stripe_dicts   = host_stripe_dicts;
    d_stripe_dicts = dev_stripe_dicts;
  }

  [[nodiscard]] auto const& host_stripe_dict(size_t stripe) const
  {
    CUDF_EXPECTS(is_string(), "Stripe dictionary is only present in string columns.");
    return stripe_dicts[stripe];
  }

  [[nodiscard]] auto const& device_stripe_dicts() const noexcept { return d_stripe_dicts; }

  // Index in the table
  [[nodiscard]] uint32_t index() const noexcept { return _index; }
  // Index in the table, including only string columns
  [[nodiscard]] uint32_t str_index() const noexcept { return _str_idx; }
  // Id in the ORC file
  [[nodiscard]] auto id() const noexcept { return _index + 1; }

  [[nodiscard]] auto is_child() const noexcept { return _is_child; }
  auto parent_index() const noexcept { return _parent_index.value(); }
  auto child_begin() const noexcept { return children.cbegin(); }
  auto child_end() const noexcept { return children.cend(); }
  auto num_children() const noexcept { return children.size(); }

  [[nodiscard]] auto type_width() const noexcept { return _type_width; }
  auto size() const noexcept { return cudf_column.size(); }

  auto null_count() const noexcept { return cudf_column.null_count(); }
  auto null_mask() const noexcept { return cudf_column.null_mask(); }
  [[nodiscard]] bool nullable() const noexcept { return null_mask() != nullptr; }
  auto user_defined_nullable() const noexcept { return nullable_from_metadata; }

  [[nodiscard]] auto scale() const noexcept { return _scale; }
  [[nodiscard]] auto precision() const noexcept { return _precision; }

  void set_orc_encoding(ColumnEncodingKind e) noexcept { _encoding_kind = e; }
  [[nodiscard]] auto orc_kind() const noexcept { return _type_kind; }
  [[nodiscard]] auto orc_encoding() const noexcept { return _encoding_kind; }
  [[nodiscard]] std::string_view orc_name() const noexcept { return name; }

 private:
  column_view cudf_column;

  // Identifier within the set of columns
  uint32_t _index = 0;
  // Identifier within the set of string columns
  int _str_idx;
  bool _is_child = false;

  // ORC-related members
  TypeKind _type_kind               = INVALID_TYPE_KIND;
  ColumnEncodingKind _encoding_kind = INVALID_ENCODING_KIND;
  std::string name;

  size_t _type_width = 0;
  int32_t _scale     = 0;
  int32_t _precision = 0;

  host_span<size_type const> rowgroup_char_counts;

  host_span<gpu::stripe_dictionary const> stripe_dicts;
  device_span<gpu::stripe_dictionary const> d_stripe_dicts;

  // Offsets for encoded decimal elements. Used to enable direct writing of encoded decimal elements
  // into the output stream.
  uint32_t* d_decimal_offsets = nullptr;

  std::optional<bool> nullable_from_metadata;
  std::vector<uint32_t> children;
  std::optional<uint32_t> _parent_index;
};

size_type orc_table_view::num_rows() const noexcept
{
  return columns.empty() ? 0 : columns.front().size();
}

namespace {
struct string_length_functor {
  __device__ inline size_type operator()(int const i) const
  {
    // we translate from 0 -> num_chunks * 2 because each statistic has a min and max
    // string and we need to calculate lengths for both.
    if (i >= num_chunks * 2) return 0;

    // min strings are even values, max strings are odd values of i
    auto const should_copy_min = i % 2 == 0;
    // index of the chunk
    auto const idx = i / 2;
    auto& str_val  = should_copy_min ? stripe_stat_chunks[idx].min_value.str_val
                                     : stripe_stat_chunks[idx].max_value.str_val;
    auto const str = stripe_stat_merge[idx].stats_dtype == dtype_string;
    return str ? str_val.length : 0;
  }

  int const num_chunks;
  statistics_chunk const* stripe_stat_chunks;
  statistics_merge_group const* stripe_stat_merge;
};

CUDF_KERNEL void copy_string_data(char* string_pool,
                                  size_type* offsets,
                                  statistics_chunk* chunks,
                                  statistics_merge_group const* groups)
{
  auto const idx = blockIdx.x / 2;
  if (groups[idx].stats_dtype == dtype_string) {
    // min strings are even values, max strings are odd values of i
    auto const should_copy_min = blockIdx.x % 2 == 0;
    auto& str_val = should_copy_min ? chunks[idx].min_value.str_val : chunks[idx].max_value.str_val;
    auto dst      = &string_pool[offsets[blockIdx.x]];
    auto src      = str_val.ptr;

    for (thread_index_type i = threadIdx.x; i < str_val.length; i += blockDim.x) {
      dst[i] = src[i];
    }
    if (threadIdx.x == 0) { str_val.ptr = dst; }
  }
}

}  // namespace

intermediate_statistics::intermediate_statistics(orc_table_view const& table,
                                                 rmm::cuda_stream_view stream)
  : stripe_stat_chunks(0, stream)
{
  std::transform(
    table.columns.begin(), table.columns.end(), std::back_inserter(col_types), [](auto const& col) {
      return col.type();
    });
}

void persisted_statistics::persist(int num_table_rows,
                                   single_write_mode write_mode,
                                   intermediate_statistics&& intermediate_stats,
                                   rmm::cuda_stream_view stream)
{
  stats_dtypes = std::move(intermediate_stats.stats_dtypes);
  col_types    = std::move(intermediate_stats.col_types);
  num_rows     = num_table_rows;
  if (num_rows == 0) { return; }

  if (write_mode == single_write_mode::NO) {
    // persist the strings in the chunks into a string pool and update pointers
    auto const num_chunks = static_cast<int>(intermediate_stats.stripe_stat_chunks.size());
    // min offset and max offset + 1 for total size
    rmm::device_uvector<size_type> offsets((num_chunks * 2) + 1, stream);

    auto iter = cudf::detail::make_counting_transform_iterator(
      0,
      string_length_functor{num_chunks,
                            intermediate_stats.stripe_stat_chunks.data(),
                            intermediate_stats.stripe_stat_merge.device_ptr()});
    thrust::exclusive_scan(rmm::exec_policy(stream), iter, iter + offsets.size(), offsets.begin());

    // pull size back to host
    auto const total_string_pool_size = offsets.element(num_chunks * 2, stream);
    if (total_string_pool_size > 0) {
      rmm::device_uvector<char> string_pool(total_string_pool_size, stream);

      // offsets describes where in the string pool each string goes. Going with the simple
      // approach for now, but it is possible something fancier with breaking up each thread into
      // copying x bytes instead of a single string is the better method since we are dealing in
      // min/max strings they almost certainly will not be uniform length.
      copy_string_data<<<num_chunks * 2, 256, 0, stream.value()>>>(
        string_pool.data(),
        offsets.data(),
        intermediate_stats.stripe_stat_chunks.data(),
        intermediate_stats.stripe_stat_merge.device_ptr());
      string_pools.emplace_back(std::move(string_pool));
    }
  }

  stripe_stat_chunks.emplace_back(std::move(intermediate_stats.stripe_stat_chunks));
  stripe_stat_merge.emplace_back(std::move(intermediate_stats.stripe_stat_merge));
}

namespace {
/**
 * @brief Gathers stripe information.
 *
 * @param columns List of columns
 * @param rowgroup_bounds Ranges of rows in each rowgroup [rowgroup][column]
 * @param max_stripe_size Maximum size of each stripe, both in bytes and in rows
 * @return List of stripe descriptors
 */
file_segmentation calculate_segmentation(host_span<orc_column_view const> columns,
                                         hostdevice_2dvector<rowgroup_rows>&& rowgroup_bounds,
                                         stripe_size_limits max_stripe_size)
{
  std::vector<stripe_rowgroups> infos;
  auto const num_rowgroups = rowgroup_bounds.size().first;
  size_t stripe_start      = 0;
  size_t stripe_bytes      = 0;
  size_type stripe_rows    = 0;
  for (size_t rg_idx = 0; rg_idx < num_rowgroups; ++rg_idx) {
    auto const rowgroup_total_bytes =
      std::accumulate(columns.begin(), columns.end(), 0ul, [&](size_t total_size, auto const& col) {
        auto const rows = rowgroup_bounds[rg_idx][col.index()].size();
        if (col.is_string()) {
          return total_size + rows + col.rowgroup_char_count(rg_idx);
        } else {
          return total_size + col.type_width() * rows;
        }
      });

    auto const rowgroup_rows_max =
      std::max_element(rowgroup_bounds[rg_idx].begin(),
                       rowgroup_bounds[rg_idx].end(),
                       [](auto& l, auto& r) { return l.size() < r.size(); })
        ->size();
    // Check if adding the current rowgroup to the stripe will make the stripe too large or long
    if ((rg_idx > stripe_start) && (stripe_bytes + rowgroup_total_bytes > max_stripe_size.bytes ||
                                    stripe_rows + rowgroup_rows_max > max_stripe_size.rows)) {
      infos.emplace_back(infos.size(), stripe_start, rg_idx - stripe_start);
      stripe_start = rg_idx;
      stripe_bytes = 0;
      stripe_rows  = 0;
    }

    stripe_bytes += rowgroup_total_bytes;
    stripe_rows += rowgroup_rows_max;
    if (rg_idx + 1 == num_rowgroups) {
      infos.emplace_back(infos.size(), stripe_start, num_rowgroups - stripe_start);
    }
  }

  return {std::move(rowgroup_bounds), std::move(infos)};
}

/**
 * @brief Returns the maximum size of RLE encoded values of an integer type.
 **/
template <typename T>
size_t max_varint_size()
{
  // varint encodes 7 bits in each byte
  return cudf::util::div_rounding_up_unsafe(sizeof(T) * 8, 7);
}

constexpr size_t RLE_stream_size(TypeKind kind, size_t count)
{
  using cudf::util::div_rounding_up_unsafe;
  constexpr auto byte_rle_max_len = 128;
  switch (kind) {
    case TypeKind::BOOLEAN:
      return div_rounding_up_unsafe(count, byte_rle_max_len * 8) * (byte_rle_max_len + 1);
    case TypeKind::BYTE:
      return div_rounding_up_unsafe(count, byte_rle_max_len) * (byte_rle_max_len + 1);
    case TypeKind::SHORT:
      return div_rounding_up_unsafe(count, gpu::encode_block_size) *
             (gpu::encode_block_size * max_varint_size<int16_t>() + 2);
    case TypeKind::FLOAT:
    case TypeKind::INT:
    case TypeKind::DATE:
      return div_rounding_up_unsafe(count, gpu::encode_block_size) *
             (gpu::encode_block_size * max_varint_size<int32_t>() + 2);
    case TypeKind::LONG:
    case TypeKind::DOUBLE:
      return div_rounding_up_unsafe(count, gpu::encode_block_size) *
             (gpu::encode_block_size * max_varint_size<int64_t>() + 2);
    default: CUDF_FAIL("Unsupported ORC type for RLE stream size");
  }
}

auto uncomp_block_alignment(CompressionKind compression_kind)
{
  if (compression_kind == NONE or
      nvcomp::is_compression_disabled(to_nvcomp_compression_type(compression_kind))) {
    return 1u;
  }

  return 1u << nvcomp::compress_input_alignment_bits(to_nvcomp_compression_type(compression_kind));
}

auto comp_block_alignment(CompressionKind compression_kind)
{
  if (compression_kind == NONE or
      nvcomp::is_compression_disabled(to_nvcomp_compression_type(compression_kind))) {
    return 1u;
  }

  return 1u << nvcomp::compress_output_alignment_bits(to_nvcomp_compression_type(compression_kind));
}

/**
 * @brief Builds up per-column streams.
 *
 * @param[in,out] columns List of columns
 * @param[in] segmentation stripe and rowgroup ranges
 * @param[in] decimal_column_sizes Sizes of encoded decimal columns
 * @return List of stream descriptors
 */
orc_streams create_streams(host_span<orc_column_view> columns,
                           file_segmentation const& segmentation,
                           std::map<uint32_t, size_t> const& decimal_column_sizes,
                           bool enable_dictionary,
                           CompressionKind compression_kind,
                           single_write_mode write_mode)
{
  // 'column 0' row index stream
  std::vector<Stream> streams{{ROW_INDEX, 0}};  // TODO: Separate index and data streams?
  // First n + 1 streams are row index streams
  streams.reserve(columns.size() + 1);
  std::transform(columns.begin(), columns.end(), std::back_inserter(streams), [](auto const& col) {
    return Stream{ROW_INDEX, col.id()};
  });

  std::vector<int32_t> ids(columns.size() * gpu::CI_NUM_STREAMS, -1);
  std::vector<TypeKind> types(streams.size(), INVALID_TYPE_KIND);

  for (auto& column : columns) {
    auto const is_nullable = [&]() -> bool {
      if (write_mode == single_write_mode::YES) {
        return column.nullable();
      } else {
        // For chunked write, when not provided nullability, we assume the worst case scenario
        // that all columns are nullable.
        auto const chunked_nullable = column.user_defined_nullable().value_or(true);
        CUDF_EXPECTS(chunked_nullable or column.null_count() == 0,
                     "Mismatch in metadata prescribed nullability and input column. "
                     "Metadata for input column with nulls cannot prescribe nullability = false");
        return chunked_nullable;
      }
    }();

    auto RLE_column_size = [&](TypeKind type_kind) {
      return std::accumulate(
        thrust::make_counting_iterator(0ul),
        thrust::make_counting_iterator(segmentation.num_rowgroups()),
        0ul,
        [&](auto data_size, auto rg_idx) {
          return data_size +
                 RLE_stream_size(type_kind, segmentation.rowgroups[rg_idx][column.index()].size());
        });
    };

    auto const kind = column.orc_kind();

    auto add_stream =
      [&](gpu::StreamIndexType index_type, StreamKind kind, TypeKind type_kind, size_t size) {
        auto const max_alignment_padding = uncomp_block_alignment(compression_kind) - 1;
        const auto base                  = column.index() * gpu::CI_NUM_STREAMS;
        ids[base + index_type]           = streams.size();
        streams.push_back(orc::Stream{
          kind,
          column.id(),
          (size == 0) ? 0 : size + max_alignment_padding * segmentation.num_rowgroups()});
        types.push_back(type_kind);
      };

    auto add_RLE_stream = [&](
                            gpu::StreamIndexType index_type, StreamKind kind, TypeKind type_kind) {
      add_stream(index_type, kind, type_kind, RLE_column_size(type_kind));
    };

    if (is_nullable) { add_RLE_stream(gpu::CI_PRESENT, PRESENT, TypeKind::BOOLEAN); }
    switch (kind) {
      case TypeKind::BOOLEAN:
      case TypeKind::BYTE:
        add_RLE_stream(gpu::CI_DATA, DATA, kind);
        column.set_orc_encoding(DIRECT);
        break;
      case TypeKind::SHORT:
      case TypeKind::INT:
      case TypeKind::LONG:
      case TypeKind::DATE:
        add_RLE_stream(gpu::CI_DATA, DATA, kind);
        column.set_orc_encoding(DIRECT_V2);
        break;
      case TypeKind::FLOAT:
      case TypeKind::DOUBLE:
        // Pass through if no nulls (no RLE encoding for floating point)
        add_stream(
          gpu::CI_DATA, DATA, kind, (column.null_count() != 0) ? RLE_column_size(kind) : 0);
        column.set_orc_encoding(DIRECT);
        break;
      case TypeKind::STRING: {
        bool enable_dict           = enable_dictionary;
        size_t dict_data_size      = 0;
        size_t dict_strings        = 0;
        size_t dict_lengths_div512 = 0;
        for (auto const& stripe : segmentation.stripes) {
          auto const sd = column.host_stripe_dict(stripe.id);
          enable_dict   = (enable_dict && sd.is_enabled);
          if (enable_dict) {
            dict_strings += sd.entry_count;
            dict_lengths_div512 += (sd.entry_count + 0x1ff) >> 9;
            dict_data_size += sd.char_count;
          }
        }

        size_t const direct_data_size = column.char_count();
        if (enable_dict) {
          uint32_t dict_bits = 0;
          for (dict_bits = 1; dict_bits < 32; dict_bits <<= 1) {
            if (dict_strings <= (1ull << dict_bits)) break;
          }
          auto const valid_count = column.size() - column.null_count();
          dict_data_size += (dict_bits * valid_count + 7) >> 3;
        }

        // Decide between direct or dictionary encoding
        if (enable_dict && dict_data_size < direct_data_size) {
          add_RLE_stream(gpu::CI_DATA, DATA, TypeKind::INT);
          add_stream(gpu::CI_DATA2, LENGTH, TypeKind::INT, dict_lengths_div512 * (512 * 4 + 2));
          add_stream(
            gpu::CI_DICTIONARY, DICTIONARY_DATA, TypeKind::CHAR, std::max(dict_data_size, 1ul));
          column.set_orc_encoding(DICTIONARY_V2);
        } else {
          add_stream(gpu::CI_DATA, DATA, TypeKind::CHAR, std::max<size_t>(direct_data_size, 1));
          add_RLE_stream(gpu::CI_DATA2, LENGTH, TypeKind::INT);
          column.set_orc_encoding(DIRECT_V2);
        }
        break;
      }
      case TypeKind::TIMESTAMP:
        add_RLE_stream(gpu::CI_DATA, DATA, TypeKind::LONG);
        add_RLE_stream(gpu::CI_DATA2, SECONDARY, TypeKind::LONG);
        column.set_orc_encoding(DIRECT_V2);
        break;
      case TypeKind::DECIMAL:
        // varint values (NO RLE)
        // data_stream_size = decimal_column_sizes.at(column.index());
        add_stream(gpu::CI_DATA, DATA, TypeKind::DECIMAL, decimal_column_sizes.at(column.index()));
        // scale stream TODO: compute exact size since all elems are equal
        add_RLE_stream(gpu::CI_DATA2, SECONDARY, TypeKind::INT);
        column.set_orc_encoding(DIRECT_V2);
        break;
      case TypeKind::LIST:
      case TypeKind::MAP:
        // no data stream, only lengths
        add_RLE_stream(gpu::CI_DATA2, LENGTH, TypeKind::INT);
        column.set_orc_encoding(DIRECT_V2);
        break;
      case TypeKind::STRUCT:
        // Only has the present stream
        break;
      default: CUDF_FAIL("Unsupported ORC type kind");
    }
  }
  return {std::move(streams), std::move(ids), std::move(types)};
}

std::vector<std::vector<rowgroup_rows>> calculate_aligned_rowgroup_bounds(
  orc_table_view const& orc_table,
  file_segmentation const& segmentation,
  rmm::cuda_stream_view stream)
{
  if (segmentation.num_rowgroups() == 0) return {};

  auto d_pd_set_counts_data = rmm::device_uvector<cudf::size_type>(
    orc_table.num_columns() * segmentation.num_rowgroups(), stream);
  auto const d_pd_set_counts = device_2dspan<cudf::size_type>{
    d_pd_set_counts_data.data(), segmentation.num_rowgroups(), orc_table.num_columns()};
  gpu::reduce_pushdown_masks(orc_table.d_columns, segmentation.rowgroups, d_pd_set_counts, stream);

  auto aligned_rgs = hostdevice_2dvector<rowgroup_rows>(
    segmentation.num_rowgroups(), orc_table.num_columns(), stream);
  CUDF_CUDA_TRY(hipMemcpyAsync(aligned_rgs.base_device_ptr(),
                                segmentation.rowgroups.base_device_ptr(),
                                aligned_rgs.count() * sizeof(rowgroup_rows),
                                hipMemcpyDefault,
                                stream.value()));
  auto const d_stripes = cudf::detail::make_device_uvector_async(
    segmentation.stripes, stream, rmm::mr::get_current_device_resource());

  // One thread per column, per stripe
  thrust::for_each_n(
    rmm::exec_policy(stream),
    thrust::make_counting_iterator(0),
    orc_table.num_columns() * segmentation.num_stripes(),
    [columns = device_span<orc_column_device_view const>{orc_table.d_columns},
     stripes = device_span<stripe_rowgroups const>{d_stripes},
     d_pd_set_counts,
     out_rowgroups = device_2dspan<rowgroup_rows>{aligned_rgs}] __device__(auto& idx) {
      uint32_t const col_idx = idx / stripes.size();
      // No alignment needed for root columns
      if (not columns[col_idx].parent_index.has_value()) return;

      auto const stripe_idx     = idx % stripes.size();
      auto const stripe         = stripes[stripe_idx];
      auto const parent_col_idx = columns[col_idx].parent_index.value();
      auto const parent_column  = columns[parent_col_idx];
      auto const stripe_end     = stripe.first + stripe.size;

      auto seek_last_borrow_rg = [&](auto rg_idx, size_type& bits_to_borrow) {
        auto curr         = rg_idx + 1;
        auto curr_rg_size = [&]() {
          return parent_column.pushdown_mask != nullptr ? d_pd_set_counts[curr][parent_col_idx]
                                                        : out_rowgroups[curr][col_idx].size();
        };
        while (curr < stripe_end and curr_rg_size() <= bits_to_borrow) {
          // All bits from rowgroup borrowed, make the rowgroup empty
          out_rowgroups[curr][col_idx].begin = out_rowgroups[curr][col_idx].end;
          bits_to_borrow -= curr_rg_size();
          ++curr;
        }
        return curr;
      };

      int previously_borrowed = 0;
      for (auto rg_idx = stripe.first; rg_idx + 1 < stripe_end; ++rg_idx) {
        auto& rg = out_rowgroups[rg_idx][col_idx];

        if (parent_column.pushdown_mask == nullptr) {
          // No pushdown mask, all null mask bits will be encoded
          // Align on rowgroup size (can be misaligned for list children)
          if (rg.size() % 8) {
            auto bits_to_borrow           = 8 - rg.size() % 8;
            auto const last_borrow_rg_idx = seek_last_borrow_rg(rg_idx, bits_to_borrow);
            if (last_borrow_rg_idx == stripe_end) {
              // Didn't find enough bits to borrow, move the rowgroup end to the stripe end
              rg.end = out_rowgroups[stripe_end - 1][col_idx].end;
              // Done with this stripe
              break;
            }
            auto& last_borrow_rg = out_rowgroups[last_borrow_rg_idx][col_idx];
            last_borrow_rg.begin += bits_to_borrow;
            rg.end = last_borrow_rg.begin;
            // Skip the rowgroups we emptied in the loop
            rg_idx = last_borrow_rg_idx - 1;
          }
        } else {
          // pushdown mask present; null mask bits w/ set pushdown mask bits will be encoded
          // Use the number of set bits in pushdown mask as size
          auto bits_to_borrow = [&]() {
            auto const parent_valid_count = d_pd_set_counts[rg_idx][parent_col_idx];
            if (parent_valid_count < previously_borrowed) {
              // Borrow to make an empty rowgroup
              return previously_borrowed - parent_valid_count;
            }
            auto const misalignment = (parent_valid_count - previously_borrowed) % 8;
            return (8 - misalignment) % 8;
          }();

          if (bits_to_borrow == 0) {
            // Didn't borrow any bits for this rowgroup
            previously_borrowed = 0;
            continue;
          }

          // Find rowgroup in which we finish the search for missing bits
          auto const last_borrow_rg_idx = seek_last_borrow_rg(rg_idx, bits_to_borrow);
          if (last_borrow_rg_idx == stripe_end) {
            // Didn't find enough bits to borrow, move the rowgroup end to the stripe end
            rg.end = out_rowgroups[stripe_end - 1][col_idx].end;
            // Done with this stripe
            break;
          }

          auto& last_borrow_rg = out_rowgroups[last_borrow_rg_idx][col_idx];
          // First row that does not need to be borrowed
          auto borrow_end = last_borrow_rg.begin;

          // Adjust the number of bits to borrow in the next iteration
          previously_borrowed = bits_to_borrow;

          // Find word in which we finish the search for missing bits (guaranteed to be available)
          while (bits_to_borrow != 0) {
            auto const mask = cudf::detail::get_mask_offset_word(
              parent_column.pushdown_mask, 0, borrow_end, borrow_end + 32);
            auto const valid_in_word = __popc(mask);

            if (valid_in_word > bits_to_borrow) break;
            bits_to_borrow -= valid_in_word;
            borrow_end += 32;
          }

          // Find the last of the missing bits (guaranteed to be available)
          while (bits_to_borrow != 0) {
            if (bit_is_set(parent_column.pushdown_mask, borrow_end)) { --bits_to_borrow; };
            ++borrow_end;
          }

          last_borrow_rg.begin = borrow_end;
          rg.end               = borrow_end;
          // Skip the rowgroups we emptied in the loop
          rg_idx = last_borrow_rg_idx - 1;
        }
      }
    });

  aligned_rgs.device_to_host_sync(stream);

  std::vector<std::vector<rowgroup_rows>> h_aligned_rgs;
  h_aligned_rgs.reserve(segmentation.num_rowgroups());
  std::transform(thrust::make_counting_iterator(0ul),
                 thrust::make_counting_iterator(segmentation.num_rowgroups()),
                 std::back_inserter(h_aligned_rgs),
                 [&](auto idx) -> std::vector<rowgroup_rows> {
                   return {aligned_rgs[idx].begin(), aligned_rgs[idx].end()};
                 });

  return h_aligned_rgs;
}

struct segmented_valid_cnt_input {
  bitmask_type const* mask;
  std::vector<size_type> indices;
};

encoded_data encode_columns(orc_table_view const& orc_table,
                            encoder_decimal_info&& dec_chunk_sizes,
                            file_segmentation const& segmentation,
                            orc_streams const& streams,
                            uint32_t uncomp_block_align,
                            rmm::cuda_stream_view stream)
{
  auto const num_columns = orc_table.num_columns();
  hostdevice_2dvector<gpu::EncChunk> chunks(num_columns, segmentation.num_rowgroups(), stream);

  auto const aligned_rowgroups = calculate_aligned_rowgroup_bounds(orc_table, segmentation, stream);

  // Initialize column chunks' descriptions
  std::map<size_type, segmented_valid_cnt_input> validity_check_inputs;

  for (auto const& column : orc_table.columns) {
    for (auto const& stripe : segmentation.stripes) {
      for (auto rg_idx_it = stripe.cbegin(); rg_idx_it < stripe.cend(); ++rg_idx_it) {
        auto const rg_idx      = *rg_idx_it;
        auto& ck               = chunks[column.index()][rg_idx];
        ck.start_row           = segmentation.rowgroups[rg_idx][column.index()].begin;
        ck.num_rows            = segmentation.rowgroups[rg_idx][column.index()].size();
        ck.null_mask_start_row = aligned_rowgroups[rg_idx][column.index()].begin;
        ck.null_mask_num_rows  = aligned_rowgroups[rg_idx][column.index()].size();
        ck.encoding_kind       = column.orc_encoding();
        ck.type_kind           = column.orc_kind();
        auto const is_str_dict =
          ck.type_kind == TypeKind::STRING and ck.encoding_kind == DICTIONARY_V2;
        ck.dict_index = is_str_dict ? column.host_stripe_dict(stripe.id).index.data() : nullptr;
        ck.dict_data_order =
          is_str_dict ? column.host_stripe_dict(stripe.id).data_order.data() : nullptr;
        ck.dtype_len = (ck.type_kind == TypeKind::STRING) ? 1 : column.type_width();
        ck.scale     = column.scale();
        ck.decimal_offsets =
          (ck.type_kind == TypeKind::DECIMAL) ? column.decimal_offsets() : nullptr;
      }
    }
  }
  chunks.host_to_device_async(stream);
  // TODO (future): pass columns separately from chunks (to skip this step)
  // and remove info from chunks that is common for the entire column
  thrust::for_each_n(
    rmm::exec_policy(stream),
    thrust::make_counting_iterator(0ul),
    chunks.count(),
    [chunks = device_2dspan<gpu::EncChunk>{chunks},
     cols = device_span<orc_column_device_view const>{orc_table.d_columns}] __device__(auto& idx) {
      auto const col_idx             = idx / chunks.size().second;
      auto const rg_idx              = idx % chunks.size().second;
      chunks[col_idx][rg_idx].column = &cols[col_idx];
    });

  auto validity_check_indices = [&](size_t col_idx) {
    std::vector<size_type> indices;
    for (auto const& stripe : segmentation.stripes) {
      for (auto rg_idx_it = stripe.cbegin(); rg_idx_it < stripe.cend() - 1; ++rg_idx_it) {
        auto const& chunk = chunks[col_idx][*rg_idx_it];
        indices.push_back(chunk.start_row);
        indices.push_back(chunk.start_row + chunk.num_rows);
      }
    }
    return indices;
  };
  for (auto const& column : orc_table.columns) {
    if (column.orc_kind() == TypeKind::BOOLEAN && column.nullable()) {
      validity_check_inputs[column.index()] = {column.null_mask(),
                                               validity_check_indices(column.index())};
    }
  }
  for (auto& cnt_in : validity_check_inputs) {
    auto const valid_counts =
      cudf::detail::segmented_valid_count(cnt_in.second.mask, cnt_in.second.indices, stream);
    CUDF_EXPECTS(
      std::none_of(valid_counts.cbegin(),
                   valid_counts.cend(),
                   [](auto valid_count) { return valid_count % 8; }),
      "There's currently a bug in encoding boolean columns. Suggested workaround is to convert "
      "to int8 type."
      " Please see https://github.com/rapidsai/cudf/issues/6763 for more information.");
  }

  hostdevice_2dvector<gpu::encoder_chunk_streams> chunk_streams(
    num_columns, segmentation.num_rowgroups(), stream);
  // per-stripe, per-stream owning buffers
  std::vector<std::vector<rmm::device_uvector<uint8_t>>> encoded_data(segmentation.num_stripes());
  for (auto const& stripe : segmentation.stripes) {
    std::generate_n(std::back_inserter(encoded_data[stripe.id]), streams.size(), [stream]() {
      return rmm::device_uvector<uint8_t>(0, stream);
    });

    for (size_t col_idx = 0; col_idx < num_columns; col_idx++) {
      for (int strm_type = 0; strm_type < gpu::CI_NUM_STREAMS; ++strm_type) {
        auto const& column = orc_table.column(col_idx);
        auto col_streams   = chunk_streams[col_idx];
        auto const strm_id = streams.id(col_idx * gpu::CI_NUM_STREAMS + strm_type);

        std::for_each(stripe.cbegin(), stripe.cend(), [&](auto rg_idx) {
          col_streams[rg_idx].ids[strm_type]     = strm_id;
          col_streams[rg_idx].lengths[strm_type] = 0;
        });

        // Calculate rowgroup sizes and stripe size
        if (strm_id >= 0) {
          size_t stripe_size = 0;
          std::for_each(stripe.cbegin(), stripe.cend(), [&](auto rg_idx) {
            auto const& ck = chunks[col_idx][rg_idx];
            auto& strm     = col_streams[rg_idx];

            if ((strm_type == gpu::CI_DICTIONARY) ||
                (strm_type == gpu::CI_DATA2 && ck.encoding_kind == DICTIONARY_V2)) {
              if (rg_idx == *stripe.cbegin()) {
                auto const stripe_dict = column.host_stripe_dict(stripe.id);
                strm.lengths[strm_type] =
                  (strm_type == gpu::CI_DICTIONARY)
                    ? stripe_dict.char_count
                    : (((stripe_dict.entry_count + 0x1ff) >> 9) * (512 * 4 + 2));
              } else {
                strm.lengths[strm_type] = 0;
              }
            } else if (strm_type == gpu::CI_DATA && ck.type_kind == TypeKind::STRING &&
                       ck.encoding_kind == DIRECT_V2) {
              strm.lengths[strm_type] = std::max(column.rowgroup_char_count(rg_idx), 1);
            } else if (strm_type == gpu::CI_DATA && streams[strm_id].length == 0 &&
                       (ck.type_kind == DOUBLE || ck.type_kind == FLOAT)) {
              // Pass-through
              strm.lengths[strm_type] = ck.num_rows * ck.dtype_len;
            } else if (ck.type_kind == DECIMAL && strm_type == gpu::CI_DATA) {
              strm.lengths[strm_type] = dec_chunk_sizes.rg_sizes.at(col_idx)[rg_idx];
            } else {
              strm.lengths[strm_type] = RLE_stream_size(streams.type(strm_id), ck.num_rows);
            }
            // Allow extra space for alignment
            stripe_size += strm.lengths[strm_type] + uncomp_block_align - 1;
          });

          encoded_data[stripe.id][strm_id] = rmm::device_uvector<uint8_t>(stripe_size, stream);
        }

        // Set offsets
        for (auto rg_idx_it = stripe.cbegin(); rg_idx_it < stripe.cend(); ++rg_idx_it) {
          auto const rg_idx = *rg_idx_it;
          auto const& ck    = chunks[col_idx][rg_idx];
          auto& strm        = col_streams[rg_idx];

          if (strm_id < 0 or (strm_type == gpu::CI_DATA && streams[strm_id].length == 0 &&
                              (ck.type_kind == DOUBLE || ck.type_kind == FLOAT))) {
            strm.data_ptrs[strm_type] = nullptr;
          } else {
            if ((strm_type == gpu::CI_DICTIONARY) ||
                (strm_type == gpu::CI_DATA2 && ck.encoding_kind == DICTIONARY_V2)) {
              strm.data_ptrs[strm_type] = encoded_data[stripe.id][strm_id].data();
            } else {
              strm.data_ptrs[strm_type] = (rg_idx_it == stripe.cbegin())
                                            ? encoded_data[stripe.id][strm_id].data()
                                            : (col_streams[rg_idx - 1].data_ptrs[strm_type] +
                                               col_streams[rg_idx - 1].lengths[strm_type]);
            }
          }
          auto const misalignment =
            reinterpret_cast<intptr_t>(strm.data_ptrs[strm_type]) % uncomp_block_align;
          if (misalignment != 0) {
            strm.data_ptrs[strm_type] += (uncomp_block_align - misalignment);
          }
        }
      }
    }
  }

  chunk_streams.host_to_device_async(stream);

  if (orc_table.num_rows() > 0) {
    if (orc_table.num_string_columns() != 0) {
      auto d_stripe_dict = orc_table.string_column(0).device_stripe_dicts();
      gpu::EncodeStripeDictionaries(d_stripe_dict.data(),
                                    orc_table.d_columns,
                                    chunks,
                                    orc_table.num_string_columns(),
                                    segmentation.num_stripes(),
                                    chunk_streams,
                                    stream);
    }

    gpu::EncodeOrcColumnData(chunks, chunk_streams, stream);
  }
  chunk_streams.device_to_host_sync(stream);

  return {std::move(encoded_data), std::move(chunk_streams)};
}

// TODO: remove StripeInformation from this function and return strm_desc instead
/**
 * @brief Returns stripe information after compacting columns' individual data
 * chunks into contiguous data streams.
 *
 * @param[in] num_index_streams Total number of index streams
 * @param[in] segmentation stripe and rowgroup ranges
 * @param[in,out] enc_data ORC per-chunk streams of encoded data
 * @param[in,out] strm_desc List of stream descriptors [stripe][data_stream]
 * @param[in] stream CUDA stream used for device memory operations and kernel launches
 * @return The stripes' information
 */
std::vector<StripeInformation> gather_stripes(size_t num_index_streams,
                                              file_segmentation const& segmentation,
                                              encoded_data* enc_data,
                                              hostdevice_2dvector<gpu::StripeStream>* strm_desc,
                                              rmm::cuda_stream_view stream)
{
  if (segmentation.num_stripes() == 0) { return {}; }

  // gathered stripes - per-stripe, per-stream (same as encoded_data.data)
  std::vector<std::vector<rmm::device_uvector<uint8_t>>> gathered_stripes(enc_data->data.size());
  for (auto& stripe_data : gathered_stripes) {
    std::generate_n(std::back_inserter(stripe_data), enc_data->data[0].size(), [&]() {
      return rmm::device_uvector<uint8_t>(0, stream);
    });
  }
  std::vector<StripeInformation> stripes(segmentation.num_stripes());
  for (auto const& stripe : segmentation.stripes) {
    for (size_t col_idx = 0; col_idx < enc_data->streams.size().first; col_idx++) {
      auto const& col_streams = (enc_data->streams)[col_idx];
      // Assign stream data of column data stream(s)
      for (int k = 0; k < gpu::CI_INDEX; k++) {
        auto const stream_id = col_streams[0].ids[k];
        if (stream_id != -1) {
          auto const actual_stripe_size = std::accumulate(
            col_streams.begin() + stripe.first,
            col_streams.begin() + stripe.first + stripe.size,
            0ul,
            [&](auto const& sum, auto const& strm) { return sum + strm.lengths[k]; });

          auto const& allocated_stripe_size = enc_data->data[stripe.id][stream_id].size();
          CUDF_EXPECTS(allocated_stripe_size >= actual_stripe_size,
                       "Internal ORC writer error: insufficient allocation size for encoded data");
          // Allocate buffers of the exact size as encoded data, smaller than the original buffers.
          // Don't copying the data to exactly sized buffer when only one chunk is present to avoid
          // performance overhead from the additional copy. When there are multiple chunks, they are
          // copied anyway, to make them contiguous (i.e. gather them).
          if (stripe.size > 1 and allocated_stripe_size > actual_stripe_size) {
            gathered_stripes[stripe.id][stream_id] =
              rmm::device_uvector<uint8_t>(actual_stripe_size, stream);
          }

          auto* ss           = &(*strm_desc)[stripe.id][stream_id - num_index_streams];
          ss->data_ptr       = gathered_stripes[stripe.id][stream_id].data();
          ss->stream_size    = actual_stripe_size;
          ss->first_chunk_id = stripe.first;
          ss->num_chunks     = stripe.size;
          ss->column_id      = col_idx;
          ss->stream_type    = k;
        }
      }
    }

    stripes[stripe.id].numberOfRows =
      stripe.size == 0 ? 0
                       : segmentation.rowgroups[stripe.first + stripe.size - 1][0].end -
                           segmentation.rowgroups[stripe.first][0].begin;
  }

  strm_desc->host_to_device_async(stream);
  // TODO: use hipcub::DeviceMemcpy::Batched
  gpu::CompactOrcDataStreams(*strm_desc, enc_data->streams, stream);
  strm_desc->device_to_host_async(stream);
  enc_data->streams.device_to_host_sync(stream);

  // move the gathered stripes to encoded_data.data for lifetime management
  for (auto stripe_id = 0ul; stripe_id < enc_data->data.size(); ++stripe_id) {
    for (auto stream_id = 0ul; stream_id < enc_data->data[0].size(); ++stream_id) {
      if (not gathered_stripes[stripe_id][stream_id].is_empty())
        enc_data->data[stripe_id][stream_id] = std::move(gathered_stripes[stripe_id][stream_id]);
    }
  }

  return stripes;
}

void set_stat_desc_leaf_cols(device_span<orc_column_device_view const> columns,
                             device_span<stats_column_desc> stat_desc,
                             rmm::cuda_stream_view stream)
{
  thrust::for_each(rmm::exec_policy(stream),
                   thrust::make_counting_iterator(0ul),
                   thrust::make_counting_iterator(stat_desc.size()),
                   [=] __device__(auto idx) { stat_desc[idx].leaf_column = &columns[idx]; });
}

cudf::detail::hostdevice_vector<uint8_t> allocate_and_encode_blobs(
  cudf::detail::hostdevice_vector<statistics_merge_group>& stats_merge_groups,
  device_span<statistics_chunk const> stat_chunks,
  int num_stat_blobs,
  rmm::cuda_stream_view stream)
{
  // figure out the buffer size needed for protobuf format
  gpu::orc_init_statistics_buffersize(
    stats_merge_groups.device_ptr(), stat_chunks.data(), num_stat_blobs, stream);
  auto max_blobs = stats_merge_groups.element(num_stat_blobs - 1, stream);

  cudf::detail::hostdevice_vector<uint8_t> blobs(max_blobs.start_chunk + max_blobs.num_chunks,
                                                 stream);
  gpu::orc_encode_statistics(blobs.device_ptr(),
                             stats_merge_groups.device_ptr(),
                             stat_chunks.data(),
                             num_stat_blobs,
                             stream);
  stats_merge_groups.device_to_host_async(stream);
  blobs.device_to_host_sync(stream);
  return blobs;
}

[[nodiscard]] statistics_dtype kind_to_stats_type(TypeKind kind)
{
  switch (kind) {
    case TypeKind::BOOLEAN: return dtype_bool;
    case TypeKind::BYTE: return dtype_int8;
    case TypeKind::SHORT: return dtype_int16;
    case TypeKind::INT: return dtype_int32;
    case TypeKind::LONG: return dtype_int64;
    case TypeKind::FLOAT: return dtype_float32;
    case TypeKind::DOUBLE: return dtype_float64;
    case TypeKind::STRING: return dtype_string;
    case TypeKind::DATE: return dtype_int32;
    case TypeKind::TIMESTAMP: return dtype_timestamp64;
    case TypeKind::DECIMAL: return dtype_decimal64;
    default: return dtype_none;
  }
}

/**
 * @brief Returns column statistics in an intermediate format.
 *
 * @param statistics_freq Frequency of statistics to be included in the output file
 * @param orc_table Table information to be written
 * @param segmentation stripe and rowgroup ranges
 * @param stream CUDA stream used for device memory operations and kernel launches
 * @return The statistic information
 */
intermediate_statistics gather_statistic_blobs(statistics_freq const stats_freq,
                                               orc_table_view const& orc_table,
                                               file_segmentation const& segmentation,
                                               rmm::cuda_stream_view stream)
{
  auto const num_rowgroup_blobs     = segmentation.rowgroups.count();
  auto const num_stripe_blobs       = segmentation.num_stripes() * orc_table.num_columns();
  auto const are_statistics_enabled = stats_freq != statistics_freq::STATISTICS_NONE;
  if (not are_statistics_enabled or num_rowgroup_blobs + num_stripe_blobs == 0) {
    return intermediate_statistics{orc_table, stream};
  }

  cudf::detail::hostdevice_vector<stats_column_desc> stat_desc(orc_table.num_columns(), stream);
  cudf::detail::hostdevice_vector<statistics_merge_group> rowgroup_merge(num_rowgroup_blobs,
                                                                         stream);
  cudf::detail::hostdevice_vector<statistics_merge_group> stripe_merge(num_stripe_blobs, stream);
  std::vector<statistics_dtype> col_stats_dtypes;
  std::vector<data_type> col_types;
  auto rowgroup_stat_merge = rowgroup_merge.host_ptr();
  auto stripe_stat_merge   = stripe_merge.host_ptr();

  for (auto const& column : orc_table.columns) {
    stats_column_desc* desc = &stat_desc[column.index()];
    desc->stats_dtype       = kind_to_stats_type(column.orc_kind());
    desc->num_rows          = column.size();
    desc->num_values        = column.size();
    if (desc->stats_dtype == dtype_timestamp64) {
      // Timestamp statistics are in milliseconds
      switch (column.scale()) {
        case 9: desc->ts_scale = 1000; break;
        case 6: desc->ts_scale = 0; break;
        case 3: desc->ts_scale = -1000; break;
        case 0: desc->ts_scale = -1000000; break;
        default: desc->ts_scale = 0; break;
      }
    } else {
      desc->ts_scale = 0;
    }
    col_stats_dtypes.push_back(desc->stats_dtype);
    col_types.push_back(column.type());
    for (auto const& stripe : segmentation.stripes) {
      auto& grp       = stripe_stat_merge[column.index() * segmentation.num_stripes() + stripe.id];
      grp.col_dtype   = column.type();
      grp.stats_dtype = desc->stats_dtype;
      grp.start_chunk =
        static_cast<uint32_t>(column.index() * segmentation.num_rowgroups() + stripe.first);
      grp.num_chunks = stripe.size;
      for (auto rg_idx_it = stripe.cbegin(); rg_idx_it != stripe.cend(); ++rg_idx_it) {
        auto& rg_grp =
          rowgroup_stat_merge[column.index() * segmentation.num_rowgroups() + *rg_idx_it];
        rg_grp.col_dtype   = column.type();
        rg_grp.stats_dtype = desc->stats_dtype;
        rg_grp.start_chunk = *rg_idx_it;
        rg_grp.num_chunks  = 1;
      }
    }
  }
  stat_desc.host_to_device_async(stream);
  rowgroup_merge.host_to_device_async(stream);
  stripe_merge.host_to_device_async(stream);
  set_stat_desc_leaf_cols(orc_table.d_columns, stat_desc, stream);

  // The rowgroup stat chunks are written out in each stripe. The stripe and file-level chunks are
  // written in the footer. To prevent persisting the rowgroup stat chunks across multiple write
  // calls in a chunked write situation, these allocations are split up so stripe data can persist
  // until the footer is written and rowgroup data can be freed after being written to the stripe.
  rmm::device_uvector<statistics_chunk> rowgroup_chunks(num_rowgroup_blobs, stream);
  rmm::device_uvector<statistics_chunk> stripe_chunks(num_stripe_blobs, stream);
  auto rowgroup_stat_chunks = rowgroup_chunks.data();
  auto stripe_stat_chunks   = stripe_chunks.data();

  rmm::device_uvector<statistics_group> rowgroup_groups(num_rowgroup_blobs, stream);
  gpu::orc_init_statistics_groups(
    rowgroup_groups.data(), stat_desc.device_ptr(), segmentation.rowgroups, stream);

  detail::calculate_group_statistics<detail::io_file_format::ORC>(
    rowgroup_chunks.data(), rowgroup_groups.data(), num_rowgroup_blobs, stream);

  detail::merge_group_statistics<detail::io_file_format::ORC>(
    stripe_stat_chunks, rowgroup_stat_chunks, stripe_merge.device_ptr(), num_stripe_blobs, stream);

  // With chunked writes, the orc table can be deallocated between write calls.
  // This forces our hand to encode row groups and stripes only in this stage and further
  // we have to persist any data from the table that we need later. The
  // minimum and maximum string inside the `str_val` structure inside `statistics_val` in
  // `statistic_chunk` that are copies of the largest and smallest strings in the row group,
  // or stripe need to be persisted between write calls. We write rowgroup data with each
  // stripe and then save each stripe's stats until the end where we merge those all together
  // to get the file-level stats.

  // Skip rowgroup blobs when encoding, if chosen granularity is coarser than "ROW_GROUP".
  auto const is_granularity_rowgroup = stats_freq == ORC_STATISTICS_ROW_GROUP;
  // we have to encode the row groups now IF they are being written out
  auto rowgroup_blobs = [&]() -> std::vector<ColStatsBlob> {
    if (not is_granularity_rowgroup) { return {}; }

    cudf::detail::hostdevice_vector<uint8_t> blobs =
      allocate_and_encode_blobs(rowgroup_merge, rowgroup_chunks, num_rowgroup_blobs, stream);

    std::vector<ColStatsBlob> rowgroup_blobs(num_rowgroup_blobs);
    for (size_t i = 0; i < num_rowgroup_blobs; i++) {
      auto const stat_begin = blobs.host_ptr(rowgroup_merge[i].start_chunk);
      auto const stat_end   = stat_begin + rowgroup_merge[i].num_chunks;
      rowgroup_blobs[i].assign(stat_begin, stat_end);
    }
    return rowgroup_blobs;
  }();

  return {std::move(rowgroup_blobs),
          std::move(stripe_chunks),
          std::move(stripe_merge),
          std::move(col_stats_dtypes),
          std::move(col_types)};
}

/**
 * @brief Returns column statistics encoded in ORC protobuf format stored in the footer.
 *
 * @param num_stripes number of stripes in the data
 * @param incoming_stats intermediate statistics returned from `gather_statistic_blobs`
 * @param stream CUDA stream used for device memory operations and kernel launches
 * @return The encoded statistic blobs
 */
encoded_footer_statistics finish_statistic_blobs(Footer const& footer,
                                                 persisted_statistics& per_chunk_stats,
                                                 rmm::cuda_stream_view stream)
{
  auto stripe_size_iter = thrust::make_transform_iterator(per_chunk_stats.stripe_stat_merge.begin(),
                                                          [](auto const& s) { return s.size(); });

  auto const num_columns = footer.types.size() - 1;
  auto const num_stripes = footer.stripes.size();

  auto const num_stripe_blobs =
    thrust::reduce(stripe_size_iter, stripe_size_iter + per_chunk_stats.stripe_stat_merge.size());
  auto const num_file_blobs = num_columns;
  auto const num_blobs      = static_cast<int>(num_stripe_blobs + num_file_blobs);

  if (num_stripe_blobs == 0) {
    if (num_file_blobs == 0) { return {}; }

    // Create empty file stats and merge groups
    std::vector<statistics_chunk> h_stat_chunks(num_file_blobs);
    cudf::detail::hostdevice_vector<statistics_merge_group> stats_merge(num_file_blobs, stream);
    // Fill in stats_merge and stat_chunks on the host
    for (auto i = 0u; i < num_file_blobs; ++i) {
      stats_merge[i].col_dtype   = per_chunk_stats.col_types[i];
      stats_merge[i].stats_dtype = kind_to_stats_type(footer.types[i + 1].kind);
      // Write the sum for empty columns, equal to zero
      h_stat_chunks[i].has_sum = true;
    }
    //  Copy to device
    auto const d_stat_chunks = cudf::detail::make_device_uvector_async<statistics_chunk>(
      h_stat_chunks, stream, rmm::mr::get_current_device_resource());
    stats_merge.host_to_device_async(stream);

    // Encode and return
    cudf::detail::hostdevice_vector<uint8_t> hd_file_blobs =
      allocate_and_encode_blobs(stats_merge, d_stat_chunks, num_file_blobs, stream);

    // Copy blobs to host (actual size)
    std::vector<ColStatsBlob> file_blobs(num_file_blobs);
    for (auto i = 0u; i < num_file_blobs; i++) {
      auto const stat_begin = hd_file_blobs.host_ptr(stats_merge[i].start_chunk);
      auto const stat_end   = stat_begin + stats_merge[i].num_chunks;
      file_blobs[i].assign(stat_begin, stat_end);
    }

    return {{}, std::move(file_blobs)};
  }

  // merge the stripe persisted data and add file data
  rmm::device_uvector<statistics_chunk> stat_chunks(num_blobs, stream);
  cudf::detail::hostdevice_vector<statistics_merge_group> stats_merge(num_blobs, stream);

  // we need to merge the stat arrays from the persisted data.
  // this needs to be done carefully because each array can contain
  // a different number of stripes and stripes from each column must be
  // located next to each other. We know the total number of stripes and
  // we know the size of each array. The number of stripes per column in a chunk array can
  // be calculated by dividing the number of chunks by the number of columns.
  // That many chunks need to be copied at a time to the proper destination.
  size_t num_entries_seen = 0;
  for (size_t i = 0; i < per_chunk_stats.stripe_stat_chunks.size(); ++i) {
    auto const stripes_per_col = per_chunk_stats.stripe_stat_chunks[i].size() / num_columns;

    auto const chunk_bytes = stripes_per_col * sizeof(statistics_chunk);
    auto const merge_bytes = stripes_per_col * sizeof(statistics_merge_group);
    for (size_t col = 0; col < num_columns; ++col) {
      CUDF_CUDA_TRY(
        hipMemcpyAsync(stat_chunks.data() + (num_stripes * col) + num_entries_seen,
                        per_chunk_stats.stripe_stat_chunks[i].data() + col * stripes_per_col,
                        chunk_bytes,
                        hipMemcpyDefault,
                        stream.value()));
      CUDF_CUDA_TRY(
        hipMemcpyAsync(stats_merge.device_ptr() + (num_stripes * col) + num_entries_seen,
                        per_chunk_stats.stripe_stat_merge[i].device_ptr() + col * stripes_per_col,
                        merge_bytes,
                        hipMemcpyDefault,
                        stream.value()));
    }
    num_entries_seen += stripes_per_col;
  }

  std::vector<statistics_merge_group> file_stats_merge(num_file_blobs);
  for (auto i = 0u; i < num_file_blobs; ++i) {
    auto col_stats         = &file_stats_merge[i];
    col_stats->col_dtype   = per_chunk_stats.col_types[i];
    col_stats->stats_dtype = per_chunk_stats.stats_dtypes[i];
    col_stats->start_chunk = static_cast<uint32_t>(i * num_stripes);
    col_stats->num_chunks  = static_cast<uint32_t>(num_stripes);
  }

  auto d_file_stats_merge = stats_merge.device_ptr(num_stripe_blobs);
  CUDF_CUDA_TRY(hipMemcpyAsync(d_file_stats_merge,
                                file_stats_merge.data(),
                                num_file_blobs * sizeof(statistics_merge_group),
                                hipMemcpyDefault,
                                stream.value()));

  auto file_stat_chunks = stat_chunks.data() + num_stripe_blobs;
  detail::merge_group_statistics<detail::io_file_format::ORC>(
    file_stat_chunks, stat_chunks.data(), d_file_stats_merge, num_file_blobs, stream);

  cudf::detail::hostdevice_vector<uint8_t> blobs =
    allocate_and_encode_blobs(stats_merge, stat_chunks, num_blobs, stream);

  auto stripe_stat_merge = stats_merge.host_ptr();

  std::vector<ColStatsBlob> stripe_blobs(num_stripe_blobs);
  for (size_t i = 0; i < num_stripe_blobs; i++) {
    auto const stat_begin = blobs.host_ptr(stripe_stat_merge[i].start_chunk);
    auto const stat_end   = stat_begin + stripe_stat_merge[i].num_chunks;
    stripe_blobs[i].assign(stat_begin, stat_end);
  }

  std::vector<ColStatsBlob> file_blobs(num_file_blobs);
  auto file_stat_merge = stats_merge.host_ptr(num_stripe_blobs);
  for (auto i = 0u; i < num_file_blobs; i++) {
    auto const stat_begin = blobs.host_ptr(file_stat_merge[i].start_chunk);
    auto const stat_end   = stat_begin + file_stat_merge[i].num_chunks;
    file_blobs[i].assign(stat_begin, stat_end);
  }

  return {std::move(stripe_blobs), std::move(file_blobs)};
}

/**
 * @brief Writes the specified column's row index stream.
 *
 * @param[in] stripe_id Stripe's identifier
 * @param[in] stream_id Stream identifier (column id + 1)
 * @param[in] columns List of columns
 * @param[in] segmentation stripe and rowgroup ranges
 * @param[in] enc_streams List of encoder chunk streams [column][rowgroup]
 * @param[in] strm_desc List of stream descriptors
 * @param[in] comp_res Output status for compressed streams
 * @param[in] rg_stats row group level statistics
 * @param[in,out] stripe Stream's parent stripe
 * @param[in,out] streams List of all streams
 * @param[in] compression_kind The compression kind
 * @param[in] compression_blocksize The block size used for compression
 * @param[in] out_sink Sink for writing data
 */
void write_index_stream(int32_t stripe_id,
                        int32_t stream_id,
                        host_span<orc_column_view const> columns,
                        file_segmentation const& segmentation,
                        host_2dspan<gpu::encoder_chunk_streams const> enc_streams,
                        host_2dspan<gpu::StripeStream const> strm_desc,
                        host_span<compression_result const> comp_res,
                        host_span<ColStatsBlob const> rg_stats,
                        StripeInformation* stripe,
                        orc_streams* streams,
                        CompressionKind compression_kind,
                        size_t compression_blocksize,
                        std::unique_ptr<data_sink> const& out_sink)
{
  row_group_index_info present;
  row_group_index_info data;
  row_group_index_info data2;
  auto const column_id = stream_id - 1;

  auto find_record = [=, &strm_desc](gpu::encoder_chunk_streams const& stream,
                                     gpu::StreamIndexType type) {
    row_group_index_info record;
    if (stream.ids[type] > 0) {
      record.pos = 0;
      if (compression_kind != NONE) {
        auto const& ss   = strm_desc[stripe_id][stream.ids[type] - (columns.size() + 1)];
        record.blk_pos   = ss.first_block;
        record.comp_pos  = 0;
        record.comp_size = ss.stream_size;
      }
    }
    return record;
  };
  auto scan_record = [=, &comp_res](gpu::encoder_chunk_streams const& stream,
                                    gpu::StreamIndexType type,
                                    row_group_index_info& record) {
    if (record.pos >= 0) {
      record.pos += stream.lengths[type];
      while ((record.pos >= 0) && (record.blk_pos >= 0) &&
             (static_cast<size_t>(record.pos) >= compression_blocksize) &&
             (record.comp_pos + block_header_size + comp_res[record.blk_pos].bytes_written <
              static_cast<size_t>(record.comp_size))) {
        record.pos -= compression_blocksize;
        record.comp_pos += block_header_size + comp_res[record.blk_pos].bytes_written;
        record.blk_pos += 1;
      }
    }
  };

  auto kind = TypeKind::STRUCT;
  // TBD: Not sure we need an empty index stream for column 0
  if (stream_id != 0) {
    auto const& strm = enc_streams[column_id][0];
    present          = find_record(strm, gpu::CI_PRESENT);
    data             = find_record(strm, gpu::CI_DATA);
    data2            = find_record(strm, gpu::CI_DATA2);

    // Change string dictionary to int from index point of view
    kind = columns[column_id].orc_kind();
    if (kind == TypeKind::STRING && columns[column_id].orc_encoding() == DICTIONARY_V2) {
      kind = TypeKind::INT;
    }
  }

  ProtobufWriter pbw((compression_kind != NONE) ? 3 : 0);

  // Add row index entries
  auto const& rowgroups_range = segmentation.stripes[stripe_id];
  std::for_each(rowgroups_range.cbegin(), rowgroups_range.cend(), [&](auto rowgroup) {
    pbw.put_row_index_entry(present.comp_pos,
                            present.pos,
                            data.comp_pos,
                            data.pos,
                            data2.comp_pos,
                            data2.pos,
                            kind,
                            (rg_stats.empty() or stream_id == 0)
                              ? nullptr
                              : (&rg_stats[column_id * segmentation.num_rowgroups() + rowgroup]));

    if (stream_id != 0) {
      const auto& strm = enc_streams[column_id][rowgroup];
      scan_record(strm, gpu::CI_PRESENT, present);
      scan_record(strm, gpu::CI_DATA, data);
      scan_record(strm, gpu::CI_DATA2, data2);
    }
  });

  (*streams)[stream_id].length = pbw.size();
  if (compression_kind != NONE) {
    uint32_t uncomp_ix_len = (uint32_t)((*streams)[stream_id].length - 3) * 2 + 1;
    pbw.buffer()[0]        = static_cast<uint8_t>(uncomp_ix_len >> 0);
    pbw.buffer()[1]        = static_cast<uint8_t>(uncomp_ix_len >> 8);
    pbw.buffer()[2]        = static_cast<uint8_t>(uncomp_ix_len >> 16);
  }
  out_sink->host_write(pbw.data(), pbw.size());
  stripe->indexLength += pbw.size();
}

/**
 * @brief Write the specified column's data streams
 *
 * @param[in] strm_desc Stream's descriptor
 * @param[in] enc_stream Chunk's streams
 * @param[in] compressed_data Compressed stream data
 * @param[in,out] stream_out Temporary host output buffer
 * @param[in,out] stripe Stream's parent stripe
 * @param[in,out] streams List of all streams
 * @param[in] compression_kind The compression kind
 * @param[in] out_sink Sink for writing data
 * @param[in] stream CUDA stream used for device memory operations and kernel launches
 * @return An std::future that should be synchronized to ensure the writing is complete
 */
std::future<void> write_data_stream(gpu::StripeStream const& strm_desc,
                                    gpu::encoder_chunk_streams const& enc_stream,
                                    uint8_t const* compressed_data,
                                    uint8_t* stream_out,
                                    StripeInformation* stripe,
                                    orc_streams* streams,
                                    CompressionKind compression_kind,
                                    std::unique_ptr<data_sink> const& out_sink,
                                    rmm::cuda_stream_view stream)
{
  auto const length                                        = strm_desc.stream_size;
  (*streams)[enc_stream.ids[strm_desc.stream_type]].length = length;
  if (length == 0) {
    return std::async(std::launch::deferred, [] {});
  }

  auto const* stream_in = (compression_kind == NONE) ? enc_stream.data_ptrs[strm_desc.stream_type]
                                                     : (compressed_data + strm_desc.bfr_offset);

  auto write_task = [&]() {
    if (out_sink->is_device_write_preferred(length)) {
      return out_sink->device_write_async(stream_in, length, stream);
    } else {
      CUDF_CUDA_TRY(
        hipMemcpyAsync(stream_out, stream_in, length, hipMemcpyDefault, stream.value()));
      stream.synchronize();

      out_sink->host_write(stream_out, length);
      return std::async(std::launch::deferred, [] {});
    }
  }();
  stripe->dataLength += length;
  return write_task;
}

/**
 * @brief Insert 3-byte uncompressed block headers in a byte vector
 *
 * @param compression_kind The compression kind
 * @param compression_blocksize The block size used for compression
 * @param v The destitation byte vector to write, which must include initial 3-byte header
 */
void add_uncompressed_block_headers(CompressionKind compression_kind,
                                    size_t compression_blocksize,
                                    std::vector<uint8_t>& v)
{
  if (compression_kind != NONE) {
    size_t uncomp_len = v.size() - 3, pos = 0, block_len;
    while (uncomp_len > compression_blocksize) {
      block_len  = compression_blocksize * 2 + 1;
      v[pos + 0] = static_cast<uint8_t>(block_len >> 0);
      v[pos + 1] = static_cast<uint8_t>(block_len >> 8);
      v[pos + 2] = static_cast<uint8_t>(block_len >> 16);
      pos += 3 + compression_blocksize;
      v.insert(v.begin() + pos, 3, 0);
      uncomp_len -= compression_blocksize;
    }
    block_len  = uncomp_len * 2 + 1;
    v[pos + 0] = static_cast<uint8_t>(block_len >> 0);
    v[pos + 1] = static_cast<uint8_t>(block_len >> 8);
    v[pos + 2] = static_cast<uint8_t>(block_len >> 16);
  }
}

void pushdown_lists_null_mask(orc_column_view const& col,
                              device_span<orc_column_device_view> d_columns,
                              bitmask_type const* parent_pd_mask,
                              device_span<bitmask_type> out_mask,
                              rmm::cuda_stream_view stream)
{
  // Set all bits - correct unless there's a mismatch between offsets and null mask
  CUDF_CUDA_TRY(hipMemsetAsync(static_cast<void*>(out_mask.data()),
                                255,
                                out_mask.size() * sizeof(bitmask_type),
                                stream.value()));

  // Reset bits where a null list element has rows in the child column
  thrust::for_each_n(
    rmm::exec_policy(stream),
    thrust::make_counting_iterator(0u),
    col.size(),
    [d_columns, col_idx = col.index(), parent_pd_mask, out_mask] __device__(auto& idx) {
      auto const d_col        = d_columns[col_idx];
      auto const is_row_valid = d_col.is_valid(idx) and bit_value_or(parent_pd_mask, idx, true);
      if (not is_row_valid) {
        auto offsets                = d_col.child(lists_column_view::offsets_column_index);
        auto const child_rows_begin = offsets.element<size_type>(idx + d_col.offset());
        auto const child_rows_end   = offsets.element<size_type>(idx + 1 + d_col.offset());
        for (auto child_row = child_rows_begin; child_row < child_rows_end; ++child_row)
          clear_bit(out_mask.data(), child_row);
      }
    });
}

/**
 * @brief All pushdown masks in a table.
 *
 * Pushdown masks are applied to child column(s). Only bits of the child column null mask that
 * correspond to set pushdown mask bits are encoded into the output file. Similarly, rows where
 * pushdown mask is 0 are treated as invalid and not included in the output.
 */
struct pushdown_null_masks {
  // Owning vector for masks in device memory
  std::vector<rmm::device_uvector<bitmask_type>> data;
  // Pointers to pushdown masks in device memory. Can be same for multiple columns.
  std::vector<bitmask_type const*> masks;
};

pushdown_null_masks init_pushdown_null_masks(orc_table_view& orc_table,
                                             rmm::cuda_stream_view stream)
{
  std::vector<bitmask_type const*> mask_ptrs;
  mask_ptrs.reserve(orc_table.num_columns());
  std::vector<rmm::device_uvector<bitmask_type>> pd_masks;
  for (auto const& col : orc_table.columns) {
    // Leaf columns don't need pushdown masks
    if (col.num_children() == 0) {
      mask_ptrs.emplace_back(nullptr);
      continue;
    }
    auto const parent_pd_mask = col.is_child() ? mask_ptrs[col.parent_index()] : nullptr;
    auto const null_mask      = col.null_mask();

    if (null_mask == nullptr and parent_pd_mask == nullptr) {
      mask_ptrs.emplace_back(nullptr);
      continue;
    }
    if (col.orc_kind() == STRUCT) {
      if (null_mask != nullptr and parent_pd_mask == nullptr) {
        // Reuse own null mask
        mask_ptrs.emplace_back(null_mask);
      } else if (null_mask == nullptr and parent_pd_mask != nullptr) {
        // Reuse parent's pushdown mask
        mask_ptrs.emplace_back(parent_pd_mask);
      } else {
        // Both are nullable, allocate new pushdown mask
        pd_masks.emplace_back(num_bitmask_words(col.size()), stream);
        mask_ptrs.emplace_back(pd_masks.back().data());

        thrust::transform(rmm::exec_policy(stream),
                          null_mask,
                          null_mask + pd_masks.back().size(),
                          parent_pd_mask,
                          pd_masks.back().data(),
                          thrust::bit_and<bitmask_type>());
      }
    }
    if (col.orc_kind() == LIST or col.orc_kind() == MAP) {
      // Need a new pushdown mask unless both the parent and current column are not nullable
      auto const child_col = orc_table.column(col.child_begin()[0]);
      // pushdown mask applies to child column(s); use the child column size
      pd_masks.emplace_back(num_bitmask_words(child_col.size()), stream);
      mask_ptrs.emplace_back(pd_masks.back().data());
      pushdown_lists_null_mask(col, orc_table.d_columns, parent_pd_mask, pd_masks.back(), stream);
    }
  }

  // Attach null masks to device column views (async)
  auto const d_mask_ptrs = cudf::detail::make_device_uvector_async(
    mask_ptrs, stream, rmm::mr::get_current_device_resource());
  thrust::for_each_n(
    rmm::exec_policy(stream),
    thrust::make_counting_iterator(0ul),
    orc_table.num_columns(),
    [cols = device_span<orc_column_device_view>{orc_table.d_columns},
     ptrs = device_span<bitmask_type const* const>{d_mask_ptrs}] __device__(auto& idx) {
      cols[idx].pushdown_mask = ptrs[idx];
    });

  return {std::move(pd_masks), std::move(mask_ptrs)};
}

template <typename T>
struct device_stack {
  __device__ device_stack(T* stack_storage, int capacity)
    : stack(stack_storage), capacity(capacity), size(0)
  {
  }
  __device__ void push(T const& val)
  {
    cudf_assert(size < capacity and "Stack overflow");
    stack[size++] = val;
  }
  __device__ T pop()
  {
    cudf_assert(size > 0 and "Stack underflow");
    return stack[--size];
  }
  __device__ bool empty() { return size == 0; }

 private:
  T* stack;
  int capacity;
  int size;
};

orc_table_view make_orc_table_view(table_view const& table,
                                   table_device_view const& d_table,
                                   table_input_metadata const& table_meta,
                                   rmm::cuda_stream_view stream)
{
  std::vector<orc_column_view> orc_columns;
  std::vector<uint32_t> str_col_indexes;

  std::function<void(column_view const&, orc_column_view*, column_in_metadata const&)>
    append_orc_column =
      [&](column_view const& col, orc_column_view* parent_col, column_in_metadata const& col_meta) {
        int const str_idx =
          (col.type().id() == type_id::STRING) ? static_cast<int>(str_col_indexes.size()) : -1;

        auto const new_col_idx = orc_columns.size();
        orc_columns.emplace_back(new_col_idx, str_idx, parent_col, col, col_meta);
        if (orc_columns[new_col_idx].is_string()) { str_col_indexes.push_back(new_col_idx); }

        auto const kind = orc_columns[new_col_idx].orc_kind();
        if (kind == TypeKind::LIST) {
          append_orc_column(col.child(lists_column_view::child_column_index),
                            &orc_columns[new_col_idx],
                            col_meta.child(lists_column_view::child_column_index));
        } else if (kind == TypeKind::STRUCT) {
          for (auto child_idx = 0; child_idx != col.num_children(); ++child_idx) {
            append_orc_column(
              col.child(child_idx), &orc_columns[new_col_idx], col_meta.child(child_idx));
          }
        } else if (kind == TypeKind::MAP) {
          // MAP: skip to the list child - include grandchildren columns instead of children
          auto const real_parent_col   = col.child(lists_column_view::child_column_index);
          auto const& real_parent_meta = col_meta.child(lists_column_view::child_column_index);
          CUDF_EXPECTS(real_parent_meta.num_children() == 2,
                       "Map struct column should have exactly two children");
          // process MAP key
          append_orc_column(
            real_parent_col.child(0), &orc_columns[new_col_idx], real_parent_meta.child(0));
          // process MAP value
          append_orc_column(
            real_parent_col.child(1), &orc_columns[new_col_idx], real_parent_meta.child(1));
        }
      };

  for (auto col_idx = 0; col_idx < table.num_columns(); ++col_idx) {
    append_orc_column(table.column(col_idx), nullptr, table_meta.column_metadata[col_idx]);
  }

  std::vector<TypeKind> type_kinds;
  type_kinds.reserve(orc_columns.size());
  std::transform(
    orc_columns.cbegin(), orc_columns.cend(), std::back_inserter(type_kinds), [](auto& orc_column) {
      return orc_column.orc_kind();
    });
  auto const d_type_kinds = cudf::detail::make_device_uvector_async(
    type_kinds, stream, rmm::mr::get_current_device_resource());

  rmm::device_uvector<orc_column_device_view> d_orc_columns(orc_columns.size(), stream);
  using stack_value_type = thrust::pair<column_device_view const*, thrust::optional<uint32_t>>;
  rmm::device_uvector<stack_value_type> stack_storage(orc_columns.size(), stream);

  // pre-order append ORC device columns
  cudf::detail::device_single_thread(
    [d_orc_cols         = device_span<orc_column_device_view>{d_orc_columns},
     d_type_kinds       = device_span<TypeKind const>{d_type_kinds},
     d_table            = d_table,
     stack_storage      = stack_storage.data(),
     stack_storage_size = stack_storage.size()] __device__() {
      device_stack stack(stack_storage, stack_storage_size);

      thrust::for_each(thrust::seq,
                       thrust::make_reverse_iterator(d_table.end()),
                       thrust::make_reverse_iterator(d_table.begin()),
                       [&stack](column_device_view const& c) {
                         stack.push({&c, thrust::nullopt});
                       });

      uint32_t idx = 0;
      while (not stack.empty()) {
        auto [col, parent] = stack.pop();
        d_orc_cols[idx]    = orc_column_device_view{*col, parent};

        if (d_type_kinds[idx] == TypeKind::MAP) {
          // Skip to the list child - do not include the child column, just grandchildren columns
          col = &col->children()[lists_column_view::child_column_index];
        }

        if (col->type().id() == type_id::LIST) {
          stack.push({&col->children()[lists_column_view::child_column_index], idx});
        } else if (col->type().id() == type_id::STRUCT) {
          thrust::for_each(thrust::seq,
                           thrust::make_reverse_iterator(col->children().end()),
                           thrust::make_reverse_iterator(col->children().begin()),
                           [&stack, idx](column_device_view const& c) {
                             stack.push({&c, idx});
                           });
        }
        ++idx;
      }
    },
    stream);

  return {std::move(orc_columns),
          std::move(d_orc_columns),
          str_col_indexes,
          cudf::detail::make_device_uvector_sync(
            str_col_indexes, stream, rmm::mr::get_current_device_resource())};
}

hostdevice_2dvector<rowgroup_rows> calculate_rowgroup_bounds(orc_table_view const& orc_table,
                                                             size_type rowgroup_size,
                                                             rmm::cuda_stream_view stream)
{
  auto const num_rowgroups =
    cudf::util::div_rounding_up_unsafe<size_t, size_t>(orc_table.num_rows(), rowgroup_size);

  hostdevice_2dvector<rowgroup_rows> rowgroup_bounds(
    num_rowgroups, orc_table.num_columns(), stream);
  thrust::for_each_n(
    rmm::exec_policy(stream),
    thrust::make_counting_iterator(0ul),
    num_rowgroups,
    [cols      = device_span<orc_column_device_view const>{orc_table.d_columns},
     rg_bounds = device_2dspan<rowgroup_rows>{rowgroup_bounds},
     rowgroup_size] __device__(auto rg_idx) mutable {
      thrust::transform(
        thrust::seq, cols.begin(), cols.end(), rg_bounds[rg_idx].begin(), [&](auto const& col) {
          // Root column
          if (!col.parent_index.has_value()) {
            size_type const rows_begin = rg_idx * rowgroup_size;
            auto const rows_end = thrust::min<size_type>((rg_idx + 1) * rowgroup_size, col.size());
            return rowgroup_rows{rows_begin, rows_end};
          } else {
            // Child column
            auto const parent_index           = *col.parent_index;
            orc_column_device_view parent_col = cols[parent_index];
            auto const parent_rg              = rg_bounds[rg_idx][parent_index];
            if (parent_col.type().id() != type_id::LIST) {
              auto const offset_diff = parent_col.offset() - col.offset();
              return rowgroup_rows{parent_rg.begin + offset_diff, parent_rg.end + offset_diff};
            }

            auto offsets = parent_col.child(lists_column_view::offsets_column_index);
            auto const rows_begin =
              offsets.element<size_type>(parent_rg.begin + parent_col.offset()) - col.offset();
            auto const rows_end =
              offsets.element<size_type>(parent_rg.end + parent_col.offset()) - col.offset();

            return rowgroup_rows{rows_begin, rows_end};
          }
        });
    });
  rowgroup_bounds.device_to_host_sync(stream);

  return rowgroup_bounds;
}

// returns host vector of per-rowgroup sizes
encoder_decimal_info decimal_chunk_sizes(orc_table_view& orc_table,
                                         file_segmentation const& segmentation,
                                         rmm::cuda_stream_view stream)
{
  std::map<uint32_t, rmm::device_uvector<uint32_t>> elem_sizes;
  // Compute per-element offsets (within each row group) on the device
  for (auto& orc_col : orc_table.columns) {
    if (orc_col.orc_kind() == DECIMAL) {
      auto& current_sizes =
        elem_sizes.insert({orc_col.index(), rmm::device_uvector<uint32_t>(orc_col.size(), stream)})
          .first->second;
      thrust::tabulate(rmm::exec_policy_nosync(stream),
                       current_sizes.begin(),
                       current_sizes.end(),
                       [d_cols  = device_span<orc_column_device_view const>{orc_table.d_columns},
                        col_idx = orc_col.index()] __device__(auto idx) {
                         auto const& col          = d_cols[col_idx];
                         auto const pushdown_mask = [&]() -> cudf::bitmask_type const* {
                           auto const parent_index = d_cols[col_idx].parent_index;
                           if (!parent_index.has_value()) return nullptr;
                           return d_cols[parent_index.value()].pushdown_mask;
                         }();

                         if (col.is_null(idx) or not bit_value_or(pushdown_mask, idx, true))
                           return 0;

                         __int128_t const element =
                           col.type().id() == type_id::DECIMAL32   ? col.element<int32_t>(idx)
                           : col.type().id() == type_id::DECIMAL64 ? col.element<int64_t>(idx)
                                                                   : col.element<__int128_t>(idx);

                         __int128_t const sign      = (element < 0) ? 1 : 0;
                         __uint128_t zigzaged_value = ((element ^ -sign) * 2) + sign;

                         return varint_size(zigzaged_value);
                       });

      orc_col.attach_decimal_offsets(current_sizes.data());
    }
  }
  if (elem_sizes.empty()) return {};

  // Compute element offsets within each row group
  gpu::decimal_sizes_to_offsets(segmentation.rowgroups, elem_sizes, stream);

  // Gather the row group sizes and copy to host
  auto d_tmp_rowgroup_sizes = rmm::device_uvector<uint32_t>(segmentation.num_rowgroups(), stream);
  std::map<uint32_t, std::vector<uint32_t>> rg_sizes;
  for (auto const& [col_idx, esizes] : elem_sizes) {
    // Copy last elem in each row group - equal to row group size
    thrust::tabulate(rmm::exec_policy(stream),
                     d_tmp_rowgroup_sizes.begin(),
                     d_tmp_rowgroup_sizes.end(),
                     [src       = esizes.data(),
                      col_idx   = col_idx,
                      rg_bounds = device_2dspan<rowgroup_rows const>{
                        segmentation.rowgroups}] __device__(auto idx) {
                       return src[rg_bounds[idx][col_idx].end - 1];
                     });

    rg_sizes[col_idx] = cudf::detail::make_std_vector_async(d_tmp_rowgroup_sizes, stream);
  }

  return {std::move(elem_sizes), std::move(rg_sizes)};
}

std::map<uint32_t, size_t> decimal_column_sizes(
  std::map<uint32_t, std::vector<uint32_t>> const& chunk_sizes)
{
  std::map<uint32_t, size_t> column_sizes;
  std::transform(chunk_sizes.cbegin(),
                 chunk_sizes.cend(),
                 std::inserter(column_sizes, column_sizes.end()),
                 [](auto const& chunk_size) -> std::pair<uint32_t, size_t> {
                   return {
                     chunk_size.first,
                     std::accumulate(chunk_size.second.cbegin(), chunk_size.second.cend(), 0lu)};
                 });
  return column_sizes;
}

size_t max_compression_output_size(CompressionKind compression_kind, uint32_t compression_blocksize)
{
  if (compression_kind == NONE) return 0;

  return compress_max_output_chunk_size(to_nvcomp_compression_type(compression_kind),
                                        compression_blocksize);
}

std::unique_ptr<table_input_metadata> make_table_meta(table_view const& input)
{
  auto table_meta = std::make_unique<table_input_metadata>(input);

  // Fill unnamed columns' names in table_meta
  std::function<void(column_in_metadata&, std::string)> add_default_name =
    [&](column_in_metadata& col_meta, std::string default_name) {
      if (col_meta.get_name().empty()) { col_meta.set_name(default_name); }
      for (size_type i = 0; i < col_meta.num_children(); ++i) {
        add_default_name(col_meta.child(i), std::to_string(i));
      }
    };
  for (size_t i = 0; i < table_meta->column_metadata.size(); ++i) {
    add_default_name(table_meta->column_metadata[i], "_col" + std::to_string(i));
  }

  return table_meta;
}

// Computes the number of characters in each rowgroup for each string column and attaches the
// results to the corresponding orc_column_view. The owning host vector is returned.
auto set_rowgroup_char_counts(orc_table_view& orc_table,
                              device_2dspan<rowgroup_rows const> rowgroup_bounds,
                              rmm::cuda_stream_view stream)
{
  auto const num_rowgroups = rowgroup_bounds.size().first;
  auto const num_str_cols  = orc_table.num_string_columns();

  auto counts         = rmm::device_uvector<size_type>(num_str_cols * num_rowgroups, stream);
  auto counts_2d_view = device_2dspan<size_type>(counts.data(), num_str_cols, num_rowgroups);
  gpu::rowgroup_char_counts(counts_2d_view,
                            orc_table.d_columns,
                            rowgroup_bounds,
                            orc_table.d_string_column_indices,
                            stream);

  auto const h_counts = cudf::detail::make_std_vector_sync(counts, stream);

  for (auto col_idx : orc_table.string_column_indices) {
    auto& str_column = orc_table.column(col_idx);
    str_column.attach_rowgroup_char_counts(
      {h_counts.data() + str_column.str_index() * num_rowgroups, num_rowgroups});
  }

  return h_counts;
}

// Holds the stripe dictionary descriptors and dictionary buffers.
struct stripe_dictionaries {
  hostdevice_2dvector<gpu::stripe_dictionary> views;       // descriptors [string_column][stripe]
  std::vector<rmm::device_uvector<uint32_t>> data_owner;   // dictionary data owner, per stripe
  std::vector<rmm::device_uvector<uint32_t>> index_owner;  // dictionary index owner, per stripe
  std::vector<rmm::device_uvector<uint32_t>> order_owner;  // dictionary order owner, per stripe

  // Should be called after encoding is complete to deallocate the dictionary buffers.
  void on_encode_complete(rmm::cuda_stream_view stream)
  {
    data_owner.clear();
    index_owner.clear();
    order_owner.clear();

    for (auto& sd : views.host_view().flat_view()) {
      sd.data       = {};
      sd.index      = {};
      sd.data_order = {};
    }
    views.host_to_device_async(stream);
  }
};

/**
 * @brief Compares two rows in a strings column
 */
struct string_rows_less {
  device_span<orc_column_device_view> cols;
  uint32_t col_idx;
  __device__ bool operator()(size_type lhs_idx, size_type rhs_idx) const
  {
    auto const& col = cols[col_idx];
    return col.element<string_view>(lhs_idx) < col.element<string_view>(rhs_idx);
  }
};

// Build stripe dictionaries for string columns
stripe_dictionaries build_dictionaries(orc_table_view& orc_table,
                                       file_segmentation const& segmentation,
                                       bool sort_dictionaries,
                                       rmm::cuda_stream_view stream)
{
  std::vector<std::vector<rmm::device_uvector<gpu::slot_type>>> hash_maps_storage(
    orc_table.string_column_indices.size());
  for (auto col_idx : orc_table.string_column_indices) {
    auto& str_column = orc_table.column(col_idx);
    for (auto const& stripe : segmentation.stripes) {
      auto const stripe_num_rows =
        stripe.size == 0 ? 0
                         : segmentation.rowgroups[stripe.first + stripe.size - 1][col_idx].end -
                             segmentation.rowgroups[stripe.first][col_idx].begin;
      hash_maps_storage[str_column.str_index()].emplace_back(stripe_num_rows * 1.43, stream);
    }
  }

  hostdevice_2dvector<gpu::stripe_dictionary> stripe_dicts(
    orc_table.num_string_columns(), segmentation.num_stripes(), stream);
  if (stripe_dicts.count() == 0) return {std::move(stripe_dicts), {}, {}};

  // Initialize stripe dictionaries
  for (auto col_idx : orc_table.string_column_indices) {
    auto& str_column       = orc_table.column(col_idx);
    auto const str_col_idx = str_column.str_index();
    str_column.attach_stripe_dicts(stripe_dicts[str_col_idx],
                                   stripe_dicts.device_view()[str_col_idx]);
    for (auto const& stripe : segmentation.stripes) {
      auto const stripe_idx = stripe.id;
      auto& sd              = stripe_dicts[str_col_idx][stripe_idx];

      sd.map_slots      = hash_maps_storage[str_col_idx][stripe_idx];
      sd.column_idx     = col_idx;
      sd.start_row      = segmentation.rowgroups[stripe.first][col_idx].begin;
      sd.start_rowgroup = stripe.first;
      sd.num_rows =
        segmentation.rowgroups[stripe.first + stripe.size - 1][col_idx].end - sd.start_row;

      sd.entry_count = 0;
      sd.char_count  = 0;
    }
  }
  stripe_dicts.host_to_device_async(stream);

  gpu::initialize_dictionary_hash_maps(stripe_dicts, stream);
  gpu::populate_dictionary_hash_maps(stripe_dicts, orc_table.d_columns, stream);
  // Copy the entry counts and char counts from the device to the host
  stripe_dicts.device_to_host_sync(stream);

  // Data owners; can be cleared after encode
  std::vector<rmm::device_uvector<uint32_t>> dict_data_owner;
  std::vector<rmm::device_uvector<uint32_t>> dict_index_owner;
  std::vector<rmm::device_uvector<uint32_t>> dict_order_owner;
  // Make decision about which stripes to encode with dictionary encoding
  for (auto col_idx : orc_table.string_column_indices) {
    auto& str_column = orc_table.column(col_idx);
    bool col_use_dictionary{false};
    for (auto const& stripe : segmentation.stripes) {
      auto const stripe_idx        = stripe.id;
      auto const str_col_idx       = str_column.str_index();
      auto& sd                     = stripe_dicts[str_col_idx][stripe_idx];
      auto const direct_char_count = std::accumulate(
        thrust::make_counting_iterator(stripe.first),
        thrust::make_counting_iterator(stripe.first + stripe.size),
        0,
        [&](auto total, auto const& rg) { return total + str_column.rowgroup_char_count(rg); });
      // Enable dictionary encoding if the dictionary size is smaller than the direct encode size
      // The estimate excludes the LENGTH stream size, which is present in both cases
      sd.is_enabled = [&]() {
        auto const dict_index_size = varint_size(sd.entry_count);
        return sd.char_count + dict_index_size * sd.entry_count < direct_char_count;
      }();
      if (sd.is_enabled) {
        dict_data_owner.emplace_back(sd.entry_count, stream);
        sd.data            = dict_data_owner.back();
        col_use_dictionary = true;
      } else {
        // Clear hash map storage as dictionary encoding is not used for this stripe
        hash_maps_storage[str_col_idx][stripe_idx] = rmm::device_uvector<gpu::slot_type>(0, stream);
        sd.map_slots                               = {};
      }
    }
    // If any stripe uses dictionary encoding, allocate index storage for the whole column
    if (col_use_dictionary) {
      dict_index_owner.emplace_back(str_column.size(), stream);
      for (auto& sd : stripe_dicts[str_column.str_index()]) {
        sd.index = dict_index_owner.back();
      }
    }
  }
  // Synchronize to ensure the copy is complete before we clear `map_slots`
  stripe_dicts.host_to_device_sync(stream);

  gpu::collect_map_entries(stripe_dicts, stream);
  gpu::get_dictionary_indices(stripe_dicts, orc_table.d_columns, stream);

  // deallocate hash map storage, unused after this point
  hash_maps_storage.clear();

  // Clear map slots and attach order buffers
  auto dictionaries_flat = stripe_dicts.host_view().flat_view();
  for (auto& sd : dictionaries_flat) {
    if (not sd.is_enabled) { continue; }

    sd.map_slots = {};
    if (sort_dictionaries) {
      dict_order_owner.emplace_back(sd.entry_count, stream);
      sd.data_order = dict_order_owner.back();
    } else {
      sd.data_order = {};
    }
  }
  stripe_dicts.host_to_device_async(stream);

  // Sort stripe dictionaries alphabetically
  if (sort_dictionaries) {
    auto streams = cudf::detail::fork_streams(stream, std::min<size_t>(dict_order_owner.size(), 8));
    auto stream_idx = 0;
    for (auto& sd : dictionaries_flat) {
      if (not sd.is_enabled) { continue; }

      auto const& current_stream = streams[stream_idx];

      // Sort the dictionary data and create a mapping from the sorted order to the original
      thrust::sequence(
        rmm::exec_policy_nosync(current_stream), sd.data_order.begin(), sd.data_order.end());
      thrust::sort_by_key(rmm::exec_policy_nosync(current_stream),
                          sd.data.begin(),
                          sd.data.end(),
                          sd.data_order.begin(),
                          string_rows_less{orc_table.d_columns, sd.column_idx});

      // Create the inverse permutation - i.e. the mapping from the original order to the sorted
      auto order_copy = cudf::detail::make_device_uvector_async<uint32_t>(
        sd.data_order, current_stream, rmm::mr::get_current_device_resource());
      thrust::scatter(rmm::exec_policy_nosync(current_stream),
                      thrust::counting_iterator<uint32_t>(0),
                      thrust::counting_iterator<uint32_t>(sd.data_order.size()),
                      order_copy.begin(),
                      sd.data_order.begin());

      stream_idx = (stream_idx + 1) % streams.size();
    }

    cudf::detail::join_streams(streams, stream);
  }

  return {std::move(stripe_dicts),
          std::move(dict_data_owner),
          std::move(dict_index_owner),
          std::move(dict_order_owner)};
}

/**
 * @brief Perform the processing steps needed to convert the input table into the output ORC data
 * for writing, such as compression and ORC encoding.
 *
 * @param input The input table
 * @param table_meta The table metadata
 * @param max_stripe_size Maximum size of stripes in the output file
 * @param row_index_stride The row index stride
 * @param enable_dictionary Whether dictionary is enabled
 * @param sort_dictionaries Whether to sort the dictionaries
 * @param compression_kind The compression kind
 * @param compression_blocksize The block size used for compression
 * @param stats_freq Column statistics granularity type for parquet/orc writers
 * @param collect_compression_stats Flag to indicate if compression statistics should be collected
 * @param write_mode Flag to indicate if there is only a single table write
 * @param out_sink Sink for writing data
 * @param stream CUDA stream used for device memory operations and kernel launches
 * @return A tuple of the intermediate results containing the processed data
 */
auto convert_table_to_orc_data(table_view const& input,
                               table_input_metadata const& table_meta,
                               stripe_size_limits max_stripe_size,
                               size_type row_index_stride,
                               bool enable_dictionary,
                               bool sort_dictionaries,
                               CompressionKind compression_kind,
                               size_t compression_blocksize,
                               statistics_freq stats_freq,
                               bool collect_compression_stats,
                               single_write_mode write_mode,
                               data_sink const& out_sink,
                               rmm::cuda_stream_view stream)
{
  auto const input_tview = table_device_view::create(input, stream);

  auto orc_table = make_orc_table_view(input, *input_tview, table_meta, stream);

  // This is unused but it holds memory buffers for later access thus needs to be kept alive.
  [[maybe_unused]] auto const pd_masks = init_pushdown_null_masks(orc_table, stream);

  auto rowgroup_bounds = calculate_rowgroup_bounds(orc_table, row_index_stride, stream);

  [[maybe_unused]] auto const rg_char_counts_data =
    set_rowgroup_char_counts(orc_table, rowgroup_bounds, stream);

  // Decide stripe boundaries based on rowgroups and char counts
  auto segmentation =
    calculate_segmentation(orc_table.columns, std::move(rowgroup_bounds), max_stripe_size);

  auto stripe_dicts    = build_dictionaries(orc_table, segmentation, sort_dictionaries, stream);
  auto dec_chunk_sizes = decimal_chunk_sizes(orc_table, segmentation, stream);

  auto const uncompressed_block_align = uncomp_block_alignment(compression_kind);
  auto const compressed_block_align   = comp_block_alignment(compression_kind);

  auto streams  = create_streams(orc_table.columns,
                                segmentation,
                                decimal_column_sizes(dec_chunk_sizes.rg_sizes),
                                enable_dictionary,
                                compression_kind,
                                write_mode);
  auto enc_data = encode_columns(
    orc_table, std::move(dec_chunk_sizes), segmentation, streams, uncompressed_block_align, stream);

  stripe_dicts.on_encode_complete(stream);

  auto const num_rows = input.num_rows();

  // Assemble individual disparate column chunks into contiguous data streams
  size_type const num_index_streams = (orc_table.num_columns() + 1);
  auto const num_data_streams       = streams.size() - num_index_streams;
  hostdevice_2dvector<gpu::StripeStream> strm_descs(
    segmentation.num_stripes(), num_data_streams, stream);
  auto stripes = gather_stripes(num_index_streams, segmentation, &enc_data, &strm_descs, stream);

  if (num_rows == 0) {
    return std::tuple{std::move(enc_data),
                      std::move(segmentation),
                      std::move(orc_table),
                      rmm::device_uvector<uint8_t>{0, stream},                // compressed_data
                      cudf::detail::hostdevice_vector<compression_result>{},  // comp_results
                      std::move(strm_descs),
                      intermediate_statistics{orc_table, stream},
                      std::optional<writer_compression_statistics>{},
                      std::move(streams),
                      std::move(stripes),
                      std::move(stripe_dicts.views),
                      cudf::detail::make_pinned_vector_async<uint8_t>(0, stream)};
  }

  // Allocate intermediate output stream buffer
  size_t compressed_bfr_size   = 0;
  size_t num_compressed_blocks = 0;

  auto const max_compressed_block_size =
    max_compression_output_size(compression_kind, compression_blocksize);
  auto const padded_max_compressed_block_size =
    util::round_up_unsafe<size_t>(max_compressed_block_size, compressed_block_align);
  auto const padded_block_header_size =
    util::round_up_unsafe<size_t>(block_header_size, compressed_block_align);

  for (auto& ss : strm_descs.host_view().flat_view()) {
    size_t stream_size = ss.stream_size;
    if (compression_kind != NONE) {
      ss.first_block = num_compressed_blocks;
      ss.bfr_offset  = compressed_bfr_size;

      auto num_blocks =
        std::max<uint32_t>((stream_size + compression_blocksize - 1) / compression_blocksize, 1);
      stream_size += num_blocks * block_header_size;
      num_compressed_blocks += num_blocks;
      compressed_bfr_size +=
        (padded_block_header_size + padded_max_compressed_block_size) * num_blocks;
    }
  }

  // Compress the data streams
  rmm::device_uvector<uint8_t> compressed_data(compressed_bfr_size, stream);
  cudf::detail::hostdevice_vector<compression_result> comp_results(num_compressed_blocks, stream);
  std::optional<writer_compression_statistics> compression_stats;
  thrust::fill(rmm::exec_policy(stream),
               comp_results.d_begin(),
               comp_results.d_end(),
               compression_result{0, compression_status::FAILURE});
  if (compression_kind != NONE) {
    strm_descs.host_to_device_async(stream);
    compression_stats = gpu::CompressOrcDataStreams(compressed_data,
                                                    num_compressed_blocks,
                                                    compression_kind,
                                                    compression_blocksize,
                                                    max_compressed_block_size,
                                                    compressed_block_align,
                                                    collect_compression_stats,
                                                    strm_descs,
                                                    enc_data.streams,
                                                    comp_results,
                                                    stream);

    // deallocate encoded data as it is not needed anymore
    enc_data.data.clear();

    strm_descs.device_to_host_async(stream);
    comp_results.device_to_host_sync(stream);
  }

  auto const max_out_stream_size = [&]() {
    uint32_t max_stream_size = 0;
    for (auto const& ss : strm_descs.host_view().flat_view()) {
      if (!out_sink.is_device_write_preferred(ss.stream_size)) {
        max_stream_size = std::max(max_stream_size, ss.stream_size);
      }
    }
    return max_stream_size;
  }();

  auto bounce_buffer = cudf::detail::make_pinned_vector_async<uint8_t>(max_out_stream_size, stream);

  auto intermediate_stats = gather_statistic_blobs(stats_freq, orc_table, segmentation, stream);

  return std::tuple{std::move(enc_data),
                    std::move(segmentation),
                    std::move(orc_table),
                    std::move(compressed_data),
                    std::move(comp_results),
                    std::move(strm_descs),
                    std::move(intermediate_stats),
                    std::move(compression_stats),
                    std::move(streams),
                    std::move(stripes),
                    std::move(stripe_dicts.views),
                    std::move(bounce_buffer)};
}

}  // namespace

writer::impl::impl(std::unique_ptr<data_sink> sink,
                   orc_writer_options const& options,
                   single_write_mode mode,
                   rmm::cuda_stream_view stream)
  : _stream(stream),
    _max_stripe_size{options.get_stripe_size_bytes(), options.get_stripe_size_rows()},
    _row_index_stride{options.get_row_index_stride()},
    _compression_kind(to_orc_compression(options.get_compression())),
    _compression_blocksize(compression_block_size(_compression_kind)),
    _compression_statistics(options.get_compression_statistics()),
    _stats_freq(options.get_statistics_freq()),
    _sort_dictionaries{options.get_enable_dictionary_sort()},
    _single_write_mode(mode),
    _kv_meta(options.get_key_value_metadata()),
    _out_sink(std::move(sink))
{
  if (options.get_metadata()) {
    _table_meta = std::make_unique<table_input_metadata>(*options.get_metadata());
  }
}

writer::impl::impl(std::unique_ptr<data_sink> sink,
                   chunked_orc_writer_options const& options,
                   single_write_mode mode,
                   rmm::cuda_stream_view stream)
  : _stream(stream),
    _max_stripe_size{options.get_stripe_size_bytes(), options.get_stripe_size_rows()},
    _row_index_stride{options.get_row_index_stride()},
    _compression_kind(to_orc_compression(options.get_compression())),
    _compression_blocksize(compression_block_size(_compression_kind)),
    _compression_statistics(options.get_compression_statistics()),
    _stats_freq(options.get_statistics_freq()),
    _sort_dictionaries{options.get_enable_dictionary_sort()},
    _single_write_mode(mode),
    _kv_meta(options.get_key_value_metadata()),
    _out_sink(std::move(sink))
{
  if (options.get_metadata()) {
    _table_meta = std::make_unique<table_input_metadata>(*options.get_metadata());
  }
}

writer::impl::~impl() { close(); }

void writer::impl::write(table_view const& input)
{
  CUDF_EXPECTS(_state != writer_state::CLOSED, "Data has already been flushed to out and closed");

  if (not _table_meta) { _table_meta = make_table_meta(input); }

  // All kinds of memory allocation and data compressions/encoding are performed here.
  // If any error occurs, such as out-of-memory exception, the internal state of the current writer
  // is still intact.
  // Note that `out_sink_` is intentionally passed by const reference to prevent accidentally
  // writing anything to it.
  [[maybe_unused]] auto [enc_data,
                         segmentation,
                         orc_table,
                         compressed_data,
                         comp_results,
                         strm_descs,
                         intermediate_stats,
                         compression_stats,
                         streams,
                         stripes,
                         stripe_dicts, /* unused, but its data will be accessed via pointer later */
                         bounce_buffer] = [&] {
    try {
      return convert_table_to_orc_data(input,
                                       *_table_meta,
                                       _max_stripe_size,
                                       _row_index_stride,
                                       _enable_dictionary,
                                       _sort_dictionaries,
                                       _compression_kind,
                                       _compression_blocksize,
                                       _stats_freq,
                                       _compression_statistics != nullptr,
                                       _single_write_mode,
                                       *_out_sink,
                                       _stream);
    } catch (...) {  // catch any exception type
      CUDF_LOG_ERROR(
        "ORC writer encountered exception during processing. "
        "No data has been written to the sink.");
      throw;  // this throws the same exception
    }
  }();

  if (_state == writer_state::NO_DATA_WRITTEN) {
    // Write the ORC file header if this is the first write
    _out_sink->host_write(MAGIC, std::strlen(MAGIC));
  }

  // Compression/encoding were all successful. Now write the intermediate results.
  write_orc_data_to_sink(enc_data,
                         segmentation,
                         orc_table,
                         compressed_data,
                         comp_results,
                         strm_descs,
                         intermediate_stats.rowgroup_blobs,
                         streams,
                         stripes,
                         bounce_buffer);

  // Update data into the footer. This needs to be called even when num_rows==0.
  add_table_to_footer_data(orc_table, stripes);

  // Update file-level and compression statistics
  update_statistics(orc_table.num_rows(), std::move(intermediate_stats), compression_stats);

  _state = writer_state::DATA_WRITTEN;
}

void writer::impl::update_statistics(
  size_type num_rows,
  intermediate_statistics&& intermediate_stats,
  std::optional<writer_compression_statistics> const& compression_stats)
{
  _persisted_stripe_statistics.persist(
    num_rows, _single_write_mode, std::move(intermediate_stats), _stream);

  if (compression_stats.has_value() and _compression_statistics != nullptr) {
    *_compression_statistics += compression_stats.value();
  }
}

void writer::impl::write_orc_data_to_sink(encoded_data const& enc_data,
                                          file_segmentation const& segmentation,
                                          orc_table_view const& orc_table,
                                          device_span<uint8_t const> compressed_data,
                                          host_span<compression_result const> comp_results,
                                          host_2dspan<gpu::StripeStream const> strm_descs,
                                          host_span<ColStatsBlob const> rg_stats,
                                          orc_streams& streams,
                                          host_span<StripeInformation> stripes,
                                          host_span<uint8_t> bounce_buffer)
{
  if (orc_table.num_rows() == 0) { return; }

  // Write stripes
  std::vector<std::future<void>> write_tasks;
  for (size_t stripe_id = 0; stripe_id < stripes.size(); ++stripe_id) {
    auto& stripe = stripes[stripe_id];

    stripe.offset = _out_sink->bytes_written();

    // Column (skippable) index streams appear at the start of the stripe
    size_type const num_index_streams = (orc_table.num_columns() + 1);
    for (size_type stream_id = 0; stream_id < num_index_streams; ++stream_id) {
      write_index_stream(stripe_id,
                         stream_id,
                         orc_table.columns,
                         segmentation,
                         enc_data.streams,
                         strm_descs,
                         comp_results,
                         rg_stats,
                         &stripe,
                         &streams,
                         _compression_kind,
                         _compression_blocksize,
                         _out_sink);
    }

    // Column data consisting one or more separate streams
    for (auto const& strm_desc : strm_descs[stripe_id]) {
      write_tasks.push_back(write_data_stream(
        strm_desc,
        enc_data.streams[strm_desc.column_id][segmentation.stripes[stripe_id].first],
        compressed_data.data(),
        bounce_buffer.data(),
        &stripe,
        &streams,
        _compression_kind,
        _out_sink,
        _stream));
    }

    // Write stripefooter consisting of stream information
    StripeFooter sf;
    sf.streams = streams;
    sf.columns.resize(orc_table.num_columns() + 1);
    sf.columns[0].kind = DIRECT;
    for (size_t i = 1; i < sf.columns.size(); ++i) {
      sf.columns[i].kind = orc_table.column(i - 1).orc_encoding();
      sf.columns[i].dictionarySize =
        (sf.columns[i].kind == DICTIONARY_V2)
          ? orc_table.column(i - 1).host_stripe_dict(stripe_id).entry_count
          : 0;
      if (orc_table.column(i - 1).orc_kind() == TIMESTAMP) { sf.writerTimezone = "UTC"; }
    }
    ProtobufWriter pbw((_compression_kind != NONE) ? 3 : 0);
    pbw.write(sf);
    stripe.footerLength = pbw.size();
    if (_compression_kind != NONE) {
      uint32_t uncomp_sf_len = (stripe.footerLength - 3) * 2 + 1;
      pbw.buffer()[0]        = static_cast<uint8_t>(uncomp_sf_len >> 0);
      pbw.buffer()[1]        = static_cast<uint8_t>(uncomp_sf_len >> 8);
      pbw.buffer()[2]        = static_cast<uint8_t>(uncomp_sf_len >> 16);
    }
    _out_sink->host_write(pbw.data(), pbw.size());
  }
  for (auto const& task : write_tasks) {
    task.wait();
  }
}

void writer::impl::add_table_to_footer_data(orc_table_view const& orc_table,
                                            std::vector<StripeInformation>& stripes)
{
  if (_footer.headerLength == 0) {
    // First call
    _footer.headerLength   = std::strlen(MAGIC);
    _footer.writer         = cudf_writer_code;
    _footer.rowIndexStride = _row_index_stride;
    _footer.types.resize(1 + orc_table.num_columns());
    _footer.types[0].kind = STRUCT;
    for (auto const& column : orc_table.columns) {
      if (!column.is_child()) {
        _footer.types[0].subtypes.emplace_back(column.id());
        _footer.types[0].fieldNames.emplace_back(column.orc_name());
      }
    }
    for (auto const& column : orc_table.columns) {
      auto& schema_type = _footer.types[column.id()];
      schema_type.kind  = column.orc_kind();
      if (column.orc_kind() == DECIMAL) {
        schema_type.scale     = static_cast<uint32_t>(column.scale());
        schema_type.precision = column.precision();
      }
      std::transform(column.child_begin(),
                     column.child_end(),
                     std::back_inserter(schema_type.subtypes),
                     [&](auto const& child_idx) { return orc_table.column(child_idx).id(); });
      if (column.orc_kind() == STRUCT) {
        std::transform(column.child_begin(),
                       column.child_end(),
                       std::back_inserter(schema_type.fieldNames),
                       [&](auto const& child_idx) {
                         return std::string{orc_table.column(child_idx).orc_name()};
                       });
      }
    }
  } else {
    // verify the user isn't passing mismatched tables
    CUDF_EXPECTS(_footer.types.size() == 1 + orc_table.num_columns(),
                 "Mismatch in table structure between multiple calls to write");
    CUDF_EXPECTS(
      std::all_of(orc_table.columns.cbegin(),
                  orc_table.columns.cend(),
                  [&](auto const& col) { return _footer.types[col.id()].kind == col.orc_kind(); }),
      "Mismatch in column types between multiple calls to write");
  }
  _footer.stripes.insert(_footer.stripes.end(),
                         std::make_move_iterator(stripes.begin()),
                         std::make_move_iterator(stripes.end()));
  _footer.numberOfRows += orc_table.num_rows();
}

void writer::impl::close()
{
  if (_state != writer_state::DATA_WRITTEN) {
    // writer is either closed or no data has been written
    _state = writer_state::CLOSED;
    return;
  }
  PostScript ps;

  if (_stats_freq != statistics_freq::STATISTICS_NONE) {
    // Write column statistics
    auto statistics = finish_statistic_blobs(_footer, _persisted_stripe_statistics, _stream);

    // File-level statistics
    {
      _footer.statistics.reserve(_footer.types.size());
      ProtobufWriter pbw;

      // Root column: number of rows
      pbw.put_uint(encode_field_number<size_type>(1));
      pbw.put_uint(_persisted_stripe_statistics.num_rows);
      // Root column: has nulls
      pbw.put_uint(encode_field_number<size_type>(10));
      pbw.put_uint(0);
      _footer.statistics.emplace_back(pbw.release());

      // Add file stats, stored after stripe stats in `column_stats`
      _footer.statistics.insert(_footer.statistics.end(),
                                std::make_move_iterator(statistics.file_level.begin()),
                                std::make_move_iterator(statistics.file_level.end()));
    }

    // Stripe-level statistics
    if (_stats_freq == statistics_freq::STATISTICS_ROWGROUP or
        _stats_freq == statistics_freq::STATISTICS_PAGE) {
      _orc_meta.stripeStats.resize(_footer.stripes.size());
      for (size_t stripe_id = 0; stripe_id < _footer.stripes.size(); stripe_id++) {
        _orc_meta.stripeStats[stripe_id].colStats.resize(_footer.types.size());
        ProtobufWriter pbw;

        // Root column: number of rows
        pbw.put_uint(encode_field_number<size_type>(1));
        pbw.put_uint(_footer.stripes[stripe_id].numberOfRows);
        // Root column: has nulls
        pbw.put_uint(encode_field_number<size_type>(10));
        pbw.put_uint(0);
        _orc_meta.stripeStats[stripe_id].colStats[0] = pbw.release();

        for (size_t col_idx = 0; col_idx < _footer.types.size() - 1; col_idx++) {
          size_t idx = _footer.stripes.size() * col_idx + stripe_id;
          _orc_meta.stripeStats[stripe_id].colStats[1 + col_idx] =
            std::move(statistics.stripe_level[idx]);
        }
      }
    }
  }

  _persisted_stripe_statistics.clear();

  _footer.contentLength = _out_sink->bytes_written();
  std::transform(
    _kv_meta.begin(), _kv_meta.end(), std::back_inserter(_footer.metadata), [&](auto const& udata) {
      return UserMetadataItem{udata.first, udata.second};
    });

  // Write statistics metadata
  if (not _orc_meta.stripeStats.empty()) {
    ProtobufWriter pbw((_compression_kind != NONE) ? 3 : 0);
    pbw.write(_orc_meta);
    add_uncompressed_block_headers(_compression_kind, _compression_blocksize, pbw.buffer());
    ps.metadataLength = pbw.size();
    _out_sink->host_write(pbw.data(), pbw.size());
  } else {
    ps.metadataLength = 0;
  }
  ProtobufWriter pbw((_compression_kind != NONE) ? 3 : 0);
  pbw.write(_footer);
  add_uncompressed_block_headers(_compression_kind, _compression_blocksize, pbw.buffer());

  // Write postscript metadata
  ps.footerLength         = pbw.size();
  ps.compression          = _compression_kind;
  ps.compressionBlockSize = _compression_blocksize;
  ps.version              = {0, 12};  // Hive 0.12
  ps.writerVersion        = cudf_writer_version;
  ps.magic                = MAGIC;

  auto const ps_length = static_cast<uint8_t>(pbw.write(ps));
  pbw.put_byte(ps_length);
  _out_sink->host_write(pbw.data(), pbw.size());
  _out_sink->flush();

  _state = writer_state::CLOSED;
}

// Forward to implementation
writer::writer(std::unique_ptr<data_sink> sink,
               orc_writer_options const& options,
               single_write_mode mode,
               rmm::cuda_stream_view stream)
  : _impl(std::make_unique<impl>(std::move(sink), options, mode, stream))
{
}

// Forward to implementation
writer::writer(std::unique_ptr<data_sink> sink,
               chunked_orc_writer_options const& options,
               single_write_mode mode,
               rmm::cuda_stream_view stream)
  : _impl(std::make_unique<impl>(std::move(sink), options, mode, stream))
{
}

// Destructor within this translation unit
writer::~writer() = default;

// Forward to implementation
void writer::write(table_view const& table) { _impl->write(table); }

// Forward to implementation
void writer::close() { _impl->close(); }

}  // namespace cudf::io::orc::detail
