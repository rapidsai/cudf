#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

/**
 * @file writer_impl.cu
 * @brief cuDF-IO ORC writer class implementation
 */

#include "writer_impl.hpp"

#include <cudf/null_mask.hpp>
#include <cudf/strings/strings_column_view.hpp>

#include <algorithm>
#include <cstring>
#include <utility>

#include <rmm/thrust_rmm_allocator.h>
#include <rmm/device_buffer.hpp>

namespace cudf {
namespace experimental {
namespace io {
namespace detail {
namespace orc {

using namespace cudf::io::orc;
using namespace cudf::io;

namespace {

/**
 * @brief Helper for pinned host memory
 **/
template <typename T>
using pinned_buffer = std::unique_ptr<T, decltype(&hipHostFree)>;

/**
 * @brief Function that translates GDF compression to ORC compression
 **/
orc::CompressionKind to_orc_compression(
    compression_type compression) {
  switch (compression) {
    case compression_type::AUTO:
    case compression_type::SNAPPY:
      return orc::CompressionKind::SNAPPY;
    case compression_type::NONE:
      return orc::CompressionKind::NONE;
    default:
      CUDF_EXPECTS(false, "Unsupported compression type");
      return orc::CompressionKind::NONE;
  }
}

/**
 * @brief Function that translates GDF dtype to ORC datatype
 **/
constexpr orc::TypeKind to_orc_type(cudf::type_id id) {
  switch (id) {
    case cudf::type_id::INT8:
      return TypeKind::BYTE;
    case cudf::type_id::INT16:
      return TypeKind::SHORT;
    case cudf::type_id::INT32:
      return TypeKind::INT;
    case cudf::type_id::INT64:
      return TypeKind::LONG;
    case cudf::type_id::FLOAT32:
      return TypeKind::FLOAT;
    case cudf::type_id::FLOAT64:
      return TypeKind::DOUBLE;
    case cudf::type_id::BOOL8:
      return TypeKind::BOOLEAN;
    case cudf::type_id::TIMESTAMP_DAYS:
      return TypeKind::DATE;
    case cudf::type_id::TIMESTAMP_SECONDS:
    case cudf::type_id::TIMESTAMP_MICROSECONDS:
    case cudf::type_id::TIMESTAMP_MILLISECONDS:
    case cudf::type_id::TIMESTAMP_NANOSECONDS:
      return TypeKind::TIMESTAMP;
    case cudf::type_id::STRING:
      return TypeKind::STRING;
    default:
      return TypeKind::INVALID_TYPE_KIND;
  }
}

/**
 * @brief Function that translates time unit to nanoscale multiple
 **/
template <typename T>
constexpr T to_clockscale(cudf::type_id timestamp_id) {
  switch (timestamp_id) {
    case cudf::type_id::TIMESTAMP_SECONDS:
      return 9;
    case cudf::type_id::TIMESTAMP_MILLISECONDS:
      return 6;
    case cudf::type_id::TIMESTAMP_MICROSECONDS:
      return 3;
    case cudf::type_id::TIMESTAMP_NANOSECONDS:
    default:
      return 0;
  }
}

}  // namespace

/**
 * @brief Helper kernel for converting string data/offsets into nvstrdesc
 * REMOVEME: Once we eliminate the legacy readers/writers, the kernels could be
 * made to use the native offset+data layout.
 **/
__global__ void stringdata_to_nvstrdesc(gpu::nvstrdesc_s *dst, const size_type *offsets,
                        const char *strdata, const uint32_t *nulls,
                        size_type column_size) {
  size_type row = blockIdx.x * blockDim.x + threadIdx.x;
  if (row < column_size) {
    uint32_t is_valid = (nulls) ? (nulls[row >> 5] >> (row & 0x1f)) & 1 : 1;
    size_t count;
    const char *ptr;
    if (is_valid) {
      size_type cur = offsets[row];
      size_type next = offsets[row + 1];
      ptr = strdata + cur;
      count = (next > cur) ? next - cur : 0;
    }
    else {
      ptr = nullptr;
      count = 0;
    }
    dst[row].ptr = ptr;
    dst[row].count = count;
  }
}


/**
 * @brief Helper class that adds ORC-specific column info
 **/
class orc_column_view {
 public:
  /**
   * @brief Constructor that extracts out the string position + length pairs
   * for building dictionaries for string columns
   **/
  explicit orc_column_view(size_t id, size_t str_id, column_view const &col,
                           const table_metadata *metadata, hipStream_t stream)
      : _id(id),
        _str_id(str_id),
        _string_type(col.type().id() == type_id::STRING),
        _type_width(_string_type ? 0 : cudf::size_of(col.type())),
        _data_count(col.size()),
        _null_count(col.null_count()),
        _data(col.data<uint8_t>()),
        _nulls(col.has_nulls() ? col.null_mask() : nullptr),
        _clockscale(to_clockscale<uint8_t>(col.type().id())),
        _type_kind(to_orc_type(col.type().id())) {
    if (_string_type && _data_count > 0) {
      strings_column_view view{col};
      _indexes = rmm::device_buffer(_data_count * sizeof(gpu::nvstrdesc_s), stream);
      stringdata_to_nvstrdesc<<< ((_data_count-1)>>8)+1, 256, 0, stream >>>(
            reinterpret_cast<gpu::nvstrdesc_s *>(_indexes.data()),
            view.offsets().data<size_type>(), view.chars().data<char>(),
            _nulls, _data_count);
      _data = _indexes.data();
      hipStreamSynchronize(stream);
    }
    // Generating default name if name isn't present in metadata
    if (metadata && _id < metadata->column_names.size()) {
      _name = metadata->column_names[_id];
    }
    else {
      _name = "_col" + std::to_string(_id);
    }
  }

  auto is_string() const noexcept { return _string_type; }
  void set_dict_stride(size_t stride) noexcept { dict_stride = stride; }
  auto get_dict_stride() const noexcept { return dict_stride; }

  /**
   * @brief Function that associates an existing dictionary chunk allocation
   **/
  void attach_dict_chunk(gpu::DictionaryChunk *host_dict,
                         gpu::DictionaryChunk *dev_dict) {
    dict = host_dict;
    d_dict = dev_dict;
  }
  auto host_dict_chunk(size_t rowgroup) {
    assert(_string_type);
    return &dict[rowgroup * dict_stride + _str_id];
  }
  auto device_dict_chunk() const { return d_dict; }

  /**
   * @brief Function that associates an existing stripe dictionary allocation
   **/
  void attach_stripe_dict(gpu::StripeDictionary *host_stripe_dict,
                          gpu::StripeDictionary *dev_stripe_dict) {
    stripe_dict = host_stripe_dict;
    d_stripe_dict = dev_stripe_dict;
  }
  auto host_stripe_dict(size_t stripe) const {
    assert(_string_type);
    return &stripe_dict[stripe * dict_stride + _str_id];
  }
  auto device_stripe_dict() const { return d_stripe_dict; }

  size_t type_width() const noexcept { return _type_width; }
  size_t data_count() const noexcept { return _data_count; }
  size_t null_count() const noexcept { return _null_count; }
  void const *data() const noexcept { return _data; }
  uint32_t const *nulls() const noexcept { return _nulls; }
  uint8_t clockscale() const noexcept { return _clockscale; }

  void set_orc_encoding(ColumnEncodingKind e) { _encoding_kind = e; }
  auto orc_kind() const noexcept { return _type_kind; }
  auto orc_encoding() const noexcept { return _encoding_kind; }
  auto orc_name() const noexcept { return _name; }

 private:
  // Identifier within set of columns and string columns, respectively
  size_t _id = 0;
  size_t _str_id = 0;
  bool _string_type = false;

  size_t _type_width = 0;
  size_t _data_count = 0;
  size_t _null_count = 0;
  void const *_data = nullptr;
  uint32_t const *_nulls = nullptr;
  uint8_t _clockscale = 0;

  // ORC-related members
  std::string _name{};
  TypeKind _type_kind;
  ColumnEncodingKind _encoding_kind;

  // String dictionary-related members
  rmm::device_buffer _indexes;
  size_t dict_stride = 0;
  gpu::DictionaryChunk const *dict = nullptr;
  gpu::StripeDictionary const *stripe_dict = nullptr;
  gpu::DictionaryChunk *d_dict = nullptr;
  gpu::StripeDictionary *d_stripe_dict = nullptr;
};

void writer::impl::init_dictionaries(
    orc_column_view *columns, size_t num_rows,
    std::vector<int> const &str_col_ids, uint32_t *dict_data,
    uint32_t *dict_index, hostdevice_vector<gpu::DictionaryChunk> &dict,
    hipStream_t stream) {
  const size_t num_rowgroups = dict.size() / str_col_ids.size();

  // Setup per-rowgroup dictionary indexes for each dictionary-aware column
  for (size_t i = 0; i < str_col_ids.size(); ++i) {
    auto &str_column = columns[str_col_ids[i]];
    str_column.set_dict_stride(str_col_ids.size());
    str_column.attach_dict_chunk(dict.host_ptr(), dict.device_ptr());

    for (size_t g = 0; g < num_rowgroups; g++) {
      auto *ck = &dict[g * str_col_ids.size() + i];
      ck->valid_map_base = str_column.nulls();
      ck->column_data_base = str_column.data();
      ck->dict_data = dict_data + i * num_rows + g * row_index_stride_;
      ck->dict_index = dict_index + i * num_rows;  // Indexed by abs row
      ck->start_row = g * row_index_stride_;
      ck->num_rows = std::min<uint32_t>(
          row_index_stride_,
          std::max<int>(str_column.data_count() - ck->start_row, 0));
      ck->num_strings = 0;
      ck->string_char_count = 0;
      ck->num_dict_strings = 0;
      ck->dict_char_count = 0;
    }
  }

  CUDA_TRY(hipMemcpyAsync(dict.device_ptr(), dict.host_ptr(),
                           dict.memory_size(), hipMemcpyHostToDevice, stream));
  CUDA_TRY(gpu::InitDictionaryIndices(dict.device_ptr(), str_col_ids.size(),
                                      num_rowgroups, stream));
  CUDA_TRY(hipMemcpyAsync(dict.host_ptr(), dict.device_ptr(),
                           dict.memory_size(), hipMemcpyDeviceToHost, stream));
  CUDA_TRY(hipStreamSynchronize(stream));
}

void writer::impl::build_dictionaries(
    orc_column_view *columns, size_t num_rows,
    std::vector<int> const &str_col_ids,
    std::vector<uint32_t> const &stripe_list,
    hostdevice_vector<gpu::DictionaryChunk> const &dict, uint32_t *dict_index,
    hostdevice_vector<gpu::StripeDictionary> &stripe_dict,
    hipStream_t stream) {
  const auto num_rowgroups = dict.size() / str_col_ids.size();

  for (size_t i = 0; i < str_col_ids.size(); i++) {
    size_t direct_cost = 0, dict_cost = 0;
    auto &str_column = columns[str_col_ids[i]];
    str_column.attach_stripe_dict(stripe_dict.host_ptr(),
                                  stripe_dict.device_ptr());

    for (size_t j = 0, g = 0; j < stripe_list.size(); j++) {
      const auto num_chunks = stripe_list[j];
      auto *sd = &stripe_dict[j * str_col_ids.size() + i];
      sd->column_data_base = str_column.host_dict_chunk(0)->column_data_base;
      sd->dict_data = str_column.host_dict_chunk(g)->dict_data;
      sd->dict_index = dict_index + i * num_rows;  // Indexed by abs row
      sd->column_id = str_col_ids[i];
      sd->start_chunk = (uint32_t)g;
      sd->num_chunks = num_chunks;
      sd->num_strings = 0;
      sd->dict_char_count = 0;
      for (size_t k = g; k < g + num_chunks; k++) {
        const auto &dt = dict[k * str_col_ids.size() + i];
        sd->num_strings += dt.num_dict_strings;
        direct_cost += dt.string_char_count;
        dict_cost += dt.dict_char_count + dt.num_dict_strings;
      }

      g += num_chunks;
    }

    // Early disable of dictionary if it doesn't look good at the chunk level
    if (enable_dictionary_ && dict_cost >= direct_cost) {
      for (size_t j = 0; j < stripe_list.size(); j++) {
        stripe_dict[j * str_col_ids.size() + i].dict_data = nullptr;
      }
    }
  }

  CUDA_TRY(hipMemcpyAsync(stripe_dict.device_ptr(), stripe_dict.host_ptr(),
                           stripe_dict.memory_size(), hipMemcpyHostToDevice,
                           stream));
  CUDA_TRY(gpu::BuildStripeDictionaries(
      stripe_dict.device_ptr(), stripe_dict.host_ptr(), dict.device_ptr(),
      stripe_list.size(), num_rowgroups, str_col_ids.size(), stream));
  CUDA_TRY(hipMemcpyAsync(stripe_dict.host_ptr(), stripe_dict.device_ptr(),
                           stripe_dict.memory_size(), hipMemcpyDeviceToHost,
                           stream));
  CUDA_TRY(hipStreamSynchronize(stream));
}

std::vector<Stream> writer::impl::gather_streams(
    orc_column_view *columns, size_t num_columns, size_t num_rows,
    std::vector<uint32_t> const &stripe_list, std::vector<int32_t> &strm_ids) {
  // First n + 1 streams are row index streams, including 'column 0'
  std::vector<Stream> streams;
  streams.resize(num_columns + 1);
  streams[0].column = 0;
  streams[0].kind = ROW_INDEX;
  streams[0].length = 0;

  for (size_t i = 0; i < num_columns; ++i) {
    TypeKind kind = columns[i].orc_kind();
    StreamKind data_kind = DATA;
    StreamKind data2_kind = LENGTH;
    ColumnEncodingKind encoding_kind = DIRECT;

    int64_t present_stream_size = 0;
    int64_t data_stream_size = 0;
    int64_t data2_stream_size = 0;
    int64_t dict_stream_size = 0;
    if (columns[i].null_count() != 0 || columns[i].data_count() != num_rows) {
      present_stream_size = ((row_index_stride_ + 7) >> 3);
      present_stream_size += (present_stream_size + 0x7f) >> 7;
    }

    switch (kind) {
      case TypeKind::BOOLEAN:
        data_stream_size = div_rowgroups_by<int64_t>(1024) * (128 + 1);
        encoding_kind = DIRECT;
        break;
      case TypeKind::BYTE:
        data_stream_size = div_rowgroups_by<int64_t>(128) * (128 + 1);
        encoding_kind = DIRECT;
        break;
      case TypeKind::SHORT:
        data_stream_size = div_rowgroups_by<int64_t>(512) * (512 * 2 + 2);
        encoding_kind = DIRECT_V2;
        break;
      case TypeKind::FLOAT:
        // Pass through if no nulls (no RLE encoding for floating point)
        data_stream_size = (columns[i].null_count() != 0)
                               ? div_rowgroups_by<int64_t>(512) * (512 * 4 + 2)
                               : INT64_C(-1);
        encoding_kind = DIRECT;
        break;
      case TypeKind::INT:
      case TypeKind::DATE:
        data_stream_size = div_rowgroups_by<int64_t>(512) * (512 * 4 + 2);
        encoding_kind = DIRECT_V2;
        break;
      case TypeKind::DOUBLE:
        // Pass through if no nulls (no RLE encoding for floating point)
        data_stream_size = (columns[i].null_count() != 0)
                               ? div_rowgroups_by<int64_t>(512) * (512 * 8 + 2)
                               : INT64_C(-1);
        encoding_kind = DIRECT;
        break;
      case TypeKind::LONG:
        data_stream_size = div_rowgroups_by<int64_t>(512) * (512 * 8 + 2);
        encoding_kind = DIRECT_V2;
        break;
      case TypeKind::STRING: {
        bool enable_dict = enable_dictionary_;
        size_t direct_data_size = 0;
        size_t dict_data_size = 0;
        size_t dict_strings = 0;
        size_t dict_lengths_div512 = 0;
        for (size_t stripe = 0, g = 0; stripe < stripe_list.size(); stripe++) {
          const auto sd = columns[i].host_stripe_dict(stripe);
          enable_dict = (enable_dict && sd->dict_data != nullptr);
          if (enable_dict) {
            dict_strings += sd->num_strings;
            dict_lengths_div512 += (sd->num_strings + 0x1ff) >> 9;
            dict_data_size += sd->dict_char_count;
          }

          for (uint32_t k = 0; k < stripe_list[stripe]; k++, g++) {
            direct_data_size +=
                columns[i].host_dict_chunk(g)->string_char_count;
          }
        }
        if (enable_dict) {
          uint32_t dict_bits = 0;
          for (dict_bits = 1; dict_bits < 32; dict_bits <<= 1) {
            if (dict_strings <= (1ull << dict_bits)) break;
          }
          const auto valid_count =
              columns[i].data_count() - columns[i].null_count();
          dict_data_size += (dict_bits * valid_count + 7) >> 3;
        }

        // Decide between direct or dictionary encoding
        if (enable_dict && dict_data_size < direct_data_size) {
          data_stream_size = div_rowgroups_by<int64_t>(512) * (512 * 4 + 2);
          data2_stream_size = dict_lengths_div512 * (512 * 4 + 2);
          dict_stream_size = std::max<size_t>(dict_data_size, 1);
          encoding_kind = DICTIONARY_V2;
        } else {
          data_stream_size = std::max<size_t>(direct_data_size, 1);
          data2_stream_size = div_rowgroups_by<int64_t>(512) * (512 * 4 + 2);
          encoding_kind = DIRECT_V2;
        }
        break;
      }
      case TypeKind::TIMESTAMP:
        data_stream_size = ((row_index_stride_ + 0x1ff) >> 9) * (512 * 4 + 2);
        data2_stream_size = data_stream_size;
        data2_kind = SECONDARY;
        encoding_kind = DIRECT_V2;
        break;
      default:
        CUDF_FAIL("Unsupported ORC type kind");
    }

    // Initialize the column's metadata
    columns[i].set_orc_encoding(encoding_kind);

    // Initialize the column's index stream
    const auto id = static_cast<uint32_t>(1 + i);
    streams[id].column = id;
    streams[id].kind = ROW_INDEX;
    streams[id].length = 0;

    // Initialize the column's data stream(s)
    const auto base = i * gpu::CI_NUM_STREAMS;
    if (present_stream_size != 0) {
      auto len = static_cast<uint64_t>(present_stream_size);
      strm_ids[base + gpu::CI_PRESENT] = streams.size();
      streams.push_back(orc::Stream{PRESENT, id, len});
    }
    if (data_stream_size != 0) {
      auto len = static_cast<uint64_t>(std::max<int64_t>(data_stream_size, 0));
      strm_ids[base + gpu::CI_DATA] = streams.size();
      streams.push_back(orc::Stream{data_kind, id, len});
    }
    if (data2_stream_size != 0) {
      auto len = static_cast<uint64_t>(std::max<int64_t>(data2_stream_size, 0));
      strm_ids[base + gpu::CI_DATA2] = streams.size();
      streams.push_back(orc::Stream{data2_kind, id, len});
    }
    if (dict_stream_size != 0) {
      auto len = static_cast<uint64_t>(dict_stream_size);
      strm_ids[base + gpu::CI_DICTIONARY] = streams.size();
      streams.push_back(orc::Stream{DICTIONARY_DATA, id, len});
    }
  }

  return streams;
}

rmm::device_buffer writer::impl::encode_columns(
    orc_column_view *columns, size_t num_columns, size_t num_rows,
    size_t num_rowgroups, std::vector<int> const &str_col_ids,
    std::vector<uint32_t> const &stripe_list,
    std::vector<Stream> const &streams, std::vector<int32_t> const &strm_ids,
    hostdevice_vector<gpu::EncChunk> &chunks, hipStream_t stream) {
  // Allocate combined buffer for RLE data and string data output
  std::vector<size_t> strm_offsets(streams.size());
  size_t str_data_size = 0;
  auto output = [&]() {
    size_t rle_data_size = 0;
    for (size_t i = 0; i < streams.size(); ++i) {
      const auto &stream = streams[i];
      const auto &column = columns[stream.column - 1];

      if (((stream.kind == DICTIONARY_DATA || stream.kind == LENGTH) &&
           (column.orc_encoding() == DICTIONARY_V2)) ||
          ((stream.kind == DATA) && (column.orc_kind() == TypeKind::STRING &&
                                     column.orc_encoding() == DIRECT_V2))) {
        strm_offsets[i] = str_data_size;
        str_data_size += stream.length;
      } else {
        strm_offsets[i] = rle_data_size;
        rle_data_size += (stream.length * num_rowgroups + 7) & ~7;
      }
    }
    str_data_size = (str_data_size + 7) & ~7;

    return rmm::device_buffer(rle_data_size + str_data_size, stream);
  }();
  auto dst_base = static_cast<uint8_t *>(output.data());

  // Initialize column chunks' descriptions
  size_t stripe_start = 0;
  size_t stripe_id = 0;
  for (size_t j = 0; j < num_rowgroups; j++) {
    for (size_t i = 0; i < num_columns; i++) {
      auto *ck = &chunks[j * num_columns + i];
      ck->start_row = (j * row_index_stride_);
      ck->num_rows =
          std::min<uint32_t>(row_index_stride_, num_rows - ck->start_row);
      ck->valid_rows = columns[i].data_count();
      ck->encoding_kind = columns[i].orc_encoding();
      ck->type_kind = columns[i].orc_kind();
      if (ck->type_kind == TypeKind::STRING) {
        ck->valid_map_base = columns[i].nulls();
        ck->column_data_base =
            (ck->encoding_kind == DICTIONARY_V2)
                ? columns[i].host_stripe_dict(stripe_id)->dict_index
                : columns[i].data();
        ck->dtype_len = 1;
      } else {
        ck->valid_map_base = columns[i].nulls();
        ck->column_data_base = columns[i].data();
        ck->dtype_len = columns[i].type_width();
      }
      ck->scale = columns[i].clockscale();

      for (int k = 0; k < gpu::CI_NUM_STREAMS; k++) {
        const auto strm_id = strm_ids[i * gpu::CI_NUM_STREAMS + k];

        ck->strm_id[k] = strm_id;
        if (strm_id >= 0) {
          if ((k == gpu::CI_DICTIONARY) ||
              (k == gpu::CI_DATA2 && ck->encoding_kind == DICTIONARY_V2)) {
            if (j == stripe_start) {
              const int32_t dict_stride = columns[i].get_dict_stride();
              const auto stripe = columns[i].host_stripe_dict(stripe_id);
              ck->strm_len[k] =
                  (k == gpu::CI_DICTIONARY)
                      ? stripe->dict_char_count
                      : (((stripe->num_strings + 0x1ff) >> 9) * (512 * 4 + 2));
              if (stripe_id == 0) {
                ck->streams[k] = dst_base + strm_offsets[strm_id];
              } else {
                const auto *ck_up =
                    &chunks[stripe[-dict_stride].start_chunk * num_columns + i];
                ck->streams[k] = ck_up->streams[k] + ck_up->strm_len[k];
              }
            } else {
              ck->strm_len[k] = 0;
              ck->streams[k] = ck[-num_columns].streams[k];
            }
          } else if (k == gpu::CI_DATA && ck->type_kind == TypeKind::STRING &&
                     ck->encoding_kind == DIRECT_V2) {
            ck->strm_len[k] = columns[i].host_dict_chunk(j)->string_char_count;
            ck->streams[k] = (j == 0) ? dst_base + strm_offsets[strm_id]
                                      : (ck[-num_columns].streams[k] +
                                         ck[-num_columns].strm_len[k]);
          } else if (k == gpu::CI_DATA && streams[strm_id].length == 0 &&
                     (ck->type_kind == DOUBLE || ck->type_kind == FLOAT)) {
            // Pass-through
            ck->strm_len[k] = ck->num_rows * ck->dtype_len;
            ck->streams[k] = nullptr;
          } else {
            ck->strm_len[k] = streams[strm_id].length;
            ck->streams[k] = dst_base + str_data_size + strm_offsets[strm_id] +
                             streams[strm_id].length * j;
          }
        } else {
          ck->strm_len[k] = 0;
          ck->streams[k] = nullptr;
        }
      }
    }

    // Track the current stripe this rowgroup chunk belongs
    if (j + 1 == stripe_start + stripe_list[stripe_id]) {
      stripe_start = j + 1;
      stripe_id++;
    }
  }

  CUDA_TRY(hipMemcpyAsync(chunks.device_ptr(), chunks.host_ptr(),
                           chunks.memory_size(), hipMemcpyHostToDevice,
                           stream));
  if (!str_col_ids.empty()) {
    auto d_stripe_dict = columns[str_col_ids[0]].device_stripe_dict();
    CUDA_TRY(gpu::EncodeStripeDictionaries(d_stripe_dict, chunks.device_ptr(),
                                           str_col_ids.size(), num_columns,
                                           stripe_list.size(), stream));
  }
  CUDA_TRY(gpu::EncodeOrcColumnData(chunks.device_ptr(), num_columns,
                                    num_rowgroups, stream));
  CUDA_TRY(hipStreamSynchronize(stream));

  return output;
}

std::vector<StripeInformation> writer::impl::gather_stripes(
    size_t num_columns, size_t num_rows, size_t num_index_streams,
    size_t num_data_streams, std::vector<uint32_t> const &stripe_list,
    hostdevice_vector<gpu::EncChunk> &chunks,
    hostdevice_vector<gpu::StripeStream> &strm_desc, hipStream_t stream) {
  std::vector<StripeInformation> stripes(stripe_list.size());
  size_t group = 0;
  size_t stripe_start = 0;
  for (size_t s = 0; s < stripe_list.size(); s++) {
    size_t stripe_group_end = group + stripe_list[s];

    for (size_t i = 0; i < num_columns; i++) {
      const auto *ck = &chunks[group * num_columns + i];

      // Assign stream data of column data stream(s)
      for (int k = 0; k < gpu::CI_INDEX; k++) {
        const auto stream_id = ck->strm_id[k];
        if (stream_id != -1) {
          auto *ss =
              &strm_desc[s * num_data_streams + stream_id - num_index_streams];
          ss->stream_size = 0;
          ss->first_chunk_id = (group * num_columns + i);
          ss->num_chunks = (stripe_group_end - group);
          ss->column_id = i;
          ss->strm_type = k;
        }
      }
    }

    group = stripe_group_end;
    size_t stripe_end = std::min(group * row_index_stride_, num_rows);
    stripes[s].numberOfRows = stripe_end - stripe_start;
    stripe_start = stripe_end;
  }

  CUDA_TRY(hipMemcpyAsync(strm_desc.device_ptr(), strm_desc.host_ptr(),
                           strm_desc.memory_size(), hipMemcpyHostToDevice,
                           stream));
  CUDA_TRY(gpu::CompactOrcDataStreams(strm_desc.device_ptr(),
                                      chunks.device_ptr(), strm_desc.size(),
                                      num_columns, stream));
  CUDA_TRY(hipMemcpyAsync(strm_desc.host_ptr(), strm_desc.device_ptr(),
                           strm_desc.memory_size(), hipMemcpyDeviceToHost,
                           stream));
  CUDA_TRY(hipMemcpyAsync(chunks.host_ptr(), chunks.device_ptr(),
                           chunks.memory_size(), hipMemcpyDeviceToHost,
                           stream));
  CUDA_TRY(hipStreamSynchronize(stream));

  return stripes;
}

void writer::impl::write_index_stream(
    int32_t stripe_id, int32_t stream_id, orc_column_view *columns,
    size_t num_columns, size_t num_data_streams, size_t group,
    size_t groups_in_stripe, hostdevice_vector<gpu::EncChunk> const &chunks,
    hostdevice_vector<gpu::StripeStream> const &strm_desc,
    hostdevice_vector<gpu_inflate_status_s> const &comp_out,
    StripeInformation &stripe, std::vector<Stream> &streams,
    ProtobufWriter *pbw) {
  // 0: position, 1: block position, 2: compressed position, 3: compressed size
  std::array<int32_t, 4> present;
  std::array<int32_t, 4> data;
  std::array<int32_t, 4> data2;
  auto kind = TypeKind::STRUCT;

  auto find_record = [=, &strm_desc](gpu::EncChunk const &chunk,
                                     gpu::StreamIndexType type) {
    std::array<int32_t, 4> record{-1, -1, -1, -1};
    if (chunk.strm_id[type] > 0) {
      record[0] = 0;
      if (compression_kind_ != NONE) {
        const auto *ss = &strm_desc[stripe_id * num_data_streams +
                                    chunk.strm_id[type] - (num_columns + 1)];
        record[1] = ss->first_block;
        record[2] = 0;
        record[3] = ss->stream_size;
      }
    }
    return record;
  };
  auto scan_record = [=, &comp_out](gpu::EncChunk const &chunk,
                                    gpu::StreamIndexType type,
                                    std::array<int32_t, 4> &record) {
    if (record[0] >= 0) {
      record[0] += chunk.strm_len[type];
      while ((record[1] >= 0) &&
             (static_cast<size_t>(record[0]) >= compression_blocksize_) &&
             (record[3] + 3 + comp_out[record[1]].bytes_written <
              static_cast<size_t>(record[4]))) {
        record[0] -= compression_blocksize_;
        record[3] += 3 + comp_out[record[1]].bytes_written;
        record[1] += 1;
      }
    }
  };

  // TBD: Not sure we need an empty index stream for column 0
  if (stream_id != 0) {
    const auto &ck = chunks[stream_id - 1];
    present = find_record(ck, gpu::CI_PRESENT);
    data = find_record(ck, gpu::CI_DATA);
    data2 = find_record(ck, gpu::CI_DATA2);

    // Change string dictionary to int from index point of view
    kind = columns[stream_id - 1].orc_kind();
    if (kind == TypeKind::STRING &&
        columns[stream_id - 1].orc_encoding() == DICTIONARY_V2) {
      kind = TypeKind::INT;
    }
  }

  buffer_.resize((compression_kind_ != NONE) ? 3 : 0);

  // Add row index entries
  for (size_t g = group; g < group + groups_in_stripe; g++) {
    pbw->put_row_index_entry(present[2], present[0], data[2], data[0], data2[2],
                             data2[0], kind);

    if (stream_id != 0) {
      const auto &ck = chunks[g * num_columns + stream_id - 1];
      scan_record(ck, gpu::CI_PRESENT, present);
      scan_record(ck, gpu::CI_DATA, data);
      scan_record(ck, gpu::CI_DATA2, data2);
    }
  }

  streams[stream_id].length = buffer_.size();
  if (compression_kind_ != NONE) {
    uint32_t uncomp_ix_len = (uint32_t)(streams[stream_id].length - 3) * 2 + 1;
    buffer_[0] = static_cast<uint8_t>(uncomp_ix_len >> 0);
    buffer_[1] = static_cast<uint8_t>(uncomp_ix_len >> 8);
    buffer_[2] = static_cast<uint8_t>(uncomp_ix_len >> 16);
  }
  outfile_.write(reinterpret_cast<char *>(buffer_.data()), buffer_.size());
  stripe.indexLength += buffer_.size();
}

void writer::impl::write_data_stream(gpu::StripeStream const &strm_desc,
                                     gpu::EncChunk const &chunk,
                                     uint8_t const *compressed_data,
                                     uint8_t *stream_out,
                                     StripeInformation &stripe,
                                     std::vector<Stream> &streams,
                                     hipStream_t stream) {
  const auto length = strm_desc.stream_size;
  streams[chunk.strm_id[strm_desc.strm_type]].length = length;
  if (length != 0) {
    const auto *stream_in = (compression_kind_ == NONE)
                                ? chunk.streams[strm_desc.strm_type]
                                : (compressed_data + strm_desc.bfr_offset);
    CUDA_TRY(hipMemcpyAsync(stream_out, stream_in, length,
                             hipMemcpyDeviceToHost, stream));
    CUDA_TRY(hipStreamSynchronize(stream));

    outfile_.write(reinterpret_cast<char *>(stream_out), length);
  }
  stripe.dataLength += length;
}

writer::impl::impl(std::string filepath, writer_options const &options,
                   rmm::mr::device_memory_resource *mr)
    : _mr(mr) {
  compression_kind_ = to_orc_compression(options.compression);

  outfile_.open(filepath, std::ios::out | std::ios::binary | std::ios::trunc);
  CUDF_EXPECTS(outfile_.is_open(), "Cannot open output file");
}

void writer::impl::write(table_view const &table, const table_metadata *metadata, hipStream_t stream) {
  size_type num_columns = table.num_columns();
  size_type num_rows = 0;

  // Mapping of string columns for quick look-up
  std::vector<int> str_col_ids;

  // Wrapper around cudf columns to attach ORC-specific type info
  std::vector<orc_column_view> orc_columns;
  orc_columns.reserve(num_columns); // Avoids unnecessary re-allocation
  for (auto it = table.begin(); it < table.end(); ++it) {
    const auto col = *it;
    const auto current_id = orc_columns.size();
    const auto current_str_id = str_col_ids.size();

    num_rows = std::max<uint32_t>(num_rows, col.size());
    orc_columns.emplace_back(current_id, current_str_id, col, metadata, stream);
    if (orc_columns.back().is_string()) {
      str_col_ids.push_back(current_id);
    }
  }

  rmm::device_vector<uint32_t> dict_index(str_col_ids.size() * num_rows);
  rmm::device_vector<uint32_t> dict_data(str_col_ids.size() * num_rows);

  // Build per-column dictionary indices
  const auto num_rowgroups = div_by_rowgroups<size_t>(num_rows);
  const auto num_dict_chunks = num_rowgroups * str_col_ids.size();
  hostdevice_vector<gpu::DictionaryChunk> dict(num_dict_chunks);
  if (str_col_ids.size() != 0) {
    init_dictionaries(orc_columns.data(), num_rows, str_col_ids,
                      dict_data.data().get(), dict_index.data().get(), dict,
                      stream);
  }

  // Decide stripe boundaries early on, based on uncompressed size
  std::vector<uint32_t> stripe_list;
  for (size_t g = 0, stripe_start = 0, stripe_size = 0; g < num_rowgroups;
       g++) {
    size_t rowgroup_size = 0;
    for (int i = 0; i < num_columns; i++) {
      if (orc_columns[i].is_string()) {
        const auto dt = orc_columns[i].host_dict_chunk(g);
        rowgroup_size += 1 * row_index_stride_;
        rowgroup_size += dt->string_char_count;
      } else {
        rowgroup_size += orc_columns[i].type_width() * row_index_stride_;
      }
    }

    // Apply rows per stripe limit to limit string dictionaries
    const size_t max_stripe_rows = !str_col_ids.empty() ? 1000000 : 5000000;
    if ((g > stripe_start) &&
        (stripe_size + rowgroup_size > max_stripe_size_ ||
         (g + 1 - stripe_start) * row_index_stride_ > max_stripe_rows)) {
      stripe_list.push_back(g - stripe_start);
      stripe_start = g;
      stripe_size = 0;
    }
    stripe_size += rowgroup_size;
    if (g + 1 == num_rowgroups) {
      stripe_list.push_back(num_rowgroups - stripe_start);
    }
  }

  // Build stripe-level dictionaries
  const auto num_stripe_dict = stripe_list.size() * str_col_ids.size();
  hostdevice_vector<gpu::StripeDictionary> stripe_dict(num_stripe_dict);
  if (str_col_ids.size() != 0) {
    build_dictionaries(orc_columns.data(), num_rows, str_col_ids, stripe_list,
                       dict, dict_index.data().get(), stripe_dict, stream);
  }

  // Initialize streams
  std::vector<int32_t> strm_ids(num_columns * gpu::CI_NUM_STREAMS, -1);
  auto streams = gather_streams(orc_columns.data(), orc_columns.size(),
                                num_rows, stripe_list, strm_ids);

  // Encode column data chunks
  const auto num_chunks = num_rowgroups * num_columns;
  hostdevice_vector<gpu::EncChunk> chunks(num_chunks);
  auto output = encode_columns(orc_columns.data(), num_columns, num_rows,
                               num_rowgroups, str_col_ids, stripe_list, streams,
                               strm_ids, chunks, stream);

  // Assemble individual desparate column chunks into contiguous data streams
  const auto num_index_streams = (num_columns + 1);
  const auto num_data_streams = streams.size() - num_index_streams;
  const auto num_stripe_streams = stripe_list.size() * num_data_streams;
  hostdevice_vector<gpu::StripeStream> strm_desc(num_stripe_streams);
  auto stripes =
      gather_stripes(num_columns, num_rows, num_index_streams, num_data_streams,
                     stripe_list, chunks, strm_desc, stream);

  // Allocate intermediate output stream buffer
  size_t compressed_bfr_size = 0;
  size_t num_compressed_blocks = 0;
  auto stream_output = [&]() {
    size_t max_stream_size = 0;

    for (size_t stripe_id = 0; stripe_id < stripe_list.size(); stripe_id++) {
      for (size_t i = 0; i < num_data_streams; i++) {
        gpu::StripeStream *ss = &strm_desc[stripe_id * num_data_streams + i];
        size_t stream_size = ss->stream_size;
        if (compression_kind_ != NONE) {
          ss->first_block = num_compressed_blocks;
          ss->bfr_offset = compressed_bfr_size;

          auto num_blocks =
              std::max<uint32_t>((stream_size + compression_blocksize_ - 1) /
                                     compression_blocksize_,
                                 1);
          stream_size += num_blocks * 3;
          num_compressed_blocks += num_blocks;
          compressed_bfr_size += stream_size;
        }
        max_stream_size = std::max(max_stream_size, stream_size);
      }
    }

    return pinned_buffer<uint8_t>{[](size_t size) {
                                    uint8_t *ptr = nullptr;
                                    CUDA_TRY(hipHostMalloc(&ptr, size));
                                    return ptr;
                                  }(max_stream_size),
                                  hipHostFree};
  }();

  // Compress the data streams
  rmm::device_buffer compressed_data(compressed_bfr_size, stream);
  hostdevice_vector<gpu_inflate_status_s> comp_out(num_compressed_blocks);
  hostdevice_vector<gpu_inflate_input_s> comp_in(num_compressed_blocks);
  if (compression_kind_ != NONE) {
    CUDA_TRY(hipMemcpyAsync(strm_desc.device_ptr(), strm_desc.host_ptr(),
                             strm_desc.memory_size(), hipMemcpyHostToDevice,
                             stream));
    CUDA_TRY(gpu::CompressOrcDataStreams(
        static_cast<uint8_t *>(compressed_data.data()), strm_desc.device_ptr(),
        chunks.device_ptr(), comp_in.device_ptr(), comp_out.device_ptr(),
        num_stripe_streams, num_compressed_blocks, compression_kind_,
        compression_blocksize_, stream));
    CUDA_TRY(hipMemcpyAsync(strm_desc.host_ptr(), strm_desc.device_ptr(),
                             strm_desc.memory_size(), hipMemcpyDeviceToHost,
                             stream));
    CUDA_TRY(hipMemcpyAsync(comp_out.host_ptr(), comp_out.device_ptr(),
                             comp_out.memory_size(), hipMemcpyDeviceToHost,
                             stream));
    CUDA_TRY(hipStreamSynchronize(stream));
  }

  ProtobufWriter pbw_(&buffer_);

  // Write file header
  outfile_.write(MAGIC, std::strlen(MAGIC));

  // Write stripes
  size_t group = 0;
  for (size_t stripe_id = 0; stripe_id < stripes.size(); stripe_id++) {
    auto groups_in_stripe = div_by_rowgroups(stripes[stripe_id].numberOfRows);
    stripes[stripe_id].offset = outfile_.tellp();

    // Column (skippable) index streams appear at the start of the stripe
    stripes[stripe_id].indexLength = 0;
    for (size_t col_id = 0; col_id <= (size_t)num_columns; col_id++) {
      write_index_stream(stripe_id, col_id, orc_columns.data(), num_columns,
                         num_data_streams, group, groups_in_stripe, chunks,
                         strm_desc, comp_out, stripes[stripe_id], streams,
                         &pbw_);
    }

    // Column data consisting one or more separate streams
    stripes[stripe_id].dataLength = 0;
    for (size_t i = 0; i < num_data_streams; i++) {
      const auto &ss = strm_desc[stripe_id * num_data_streams + i];
      const auto &ck = chunks[group * num_columns + ss.column_id];

      write_data_stream(ss, ck, static_cast<uint8_t *>(compressed_data.data()),
                        stream_output.get(), stripes[stripe_id], streams,
                        stream);
    }

    // Write stripefooter consisting of stream information
    StripeFooter sf;
    sf.streams = streams;
    sf.columns.resize(num_columns + 1);
    sf.columns[0].kind = DIRECT;
    sf.columns[0].dictionarySize = 0;
    for (size_t i = 1; i < sf.columns.size(); ++i) {
      sf.columns[i].kind = orc_columns[i - 1].orc_encoding();
      sf.columns[i].dictionarySize =
          (sf.columns[i].kind == DICTIONARY_V2)
              ? orc_columns[i - 1].host_stripe_dict(stripe_id)->num_strings
              : 0;
      if (orc_columns[i - 1].orc_kind() == TIMESTAMP) {
        sf.writerTimezone = "UTC";
      }
    }
    buffer_.resize((compression_kind_ != NONE) ? 3 : 0);
    pbw_.write(&sf);
    stripes[stripe_id].footerLength = buffer_.size();
    if (compression_kind_ != NONE) {
      uint32_t uncomp_sf_len = (stripes[stripe_id].footerLength - 3) * 2 + 1;
      buffer_[0] = static_cast<uint8_t>(uncomp_sf_len >> 0);
      buffer_[1] = static_cast<uint8_t>(uncomp_sf_len >> 8);
      buffer_[2] = static_cast<uint8_t>(uncomp_sf_len >> 16);
    }
    outfile_.write(reinterpret_cast<char *>(buffer_.data()), buffer_.size());

    group += groups_in_stripe;
  }

  // Write filefooter metadata
  FileFooter ff;
  ff.headerLength = std::strlen(MAGIC);
  ff.contentLength = outfile_.tellp();
  ff.stripes = std::move(stripes);
  ff.numberOfRows = num_rows;
  ff.rowIndexStride = row_index_stride_;
  ff.types.resize(1 + num_columns);
  ff.types[0].kind = STRUCT;
  ff.types[0].subtypes.resize(num_columns);
  ff.types[0].fieldNames.resize(num_columns);
  for (int i = 0; i < num_columns; ++i) {
    ff.types[1 + i].kind = orc_columns[i].orc_kind();
    ff.types[0].subtypes[i] = 1 + i;
    ff.types[0].fieldNames[i] = orc_columns[i].orc_name();
  }
  if (metadata) {
    for (auto it = metadata->user_data.begin(); it != metadata->user_data.end(); it++) {
      ff.metadata.push_back({it->first, it->second});
    }
  }
  buffer_.resize((compression_kind_ != NONE) ? 3 : 0);
  pbw_.write(&ff);

  // Write postscript metadata
  PostScript ps;
  ps.footerLength = buffer_.size();
  ps.compression = compression_kind_;
  ps.compressionBlockSize = compression_blocksize_;
  ps.version = {0, 12};
  ps.metadataLength = 0;  // TODO: Write stripe statistics
  ps.magic = MAGIC;
  if (compression_kind_ != NONE) {
    // TODO: If the file footer ends up larger than the compression block
    // size, we'll need to insert additional 3-byte block headers
    uint32_t uncomp_ff_len = (uint32_t)(ps.footerLength - 3) * 2 + 1;
    buffer_[0] = static_cast<uint8_t>(uncomp_ff_len >> 0);
    buffer_[1] = static_cast<uint8_t>(uncomp_ff_len >> 8);
    buffer_[2] = static_cast<uint8_t>(uncomp_ff_len >> 16);
  }
  const auto ps_length = static_cast<uint8_t>(pbw_.write(&ps));
  buffer_.push_back(ps_length);
  outfile_.write(reinterpret_cast<char *>(buffer_.data()), buffer_.size());
  outfile_.flush();
}

// Forward to implementation
writer::writer(std::string filepath, writer_options const &options,
               rmm::mr::device_memory_resource *mr)
    : _impl(std::make_unique<impl>(filepath, options, mr)) {}

// Destructor within this translation unit
writer::~writer() = default;

// Forward to implementation
void writer::write_all(table_view const &table, const table_metadata *metadata, hipStream_t stream) {
  _impl->write(table, metadata, stream);
}

}  // namespace orc
}  // namespace detail
}  // namespace io
}  // namespace experimental
}  // namespace cudf
