#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */
#include <io/utilities/block_utils.cuh>
#include "orc_common.h"
#include "orc_gpu.h"

#include <rmm/thrust_rmm_allocator.h>

#include <thrust/device_ptr.h>
#include <thrust/execution_policy.h>
#include <thrust/sort.h>

namespace cudf {
namespace io {
namespace orc {
namespace gpu {
#define MAX_SHORT_DICT_ENTRIES (10 * 1024)
#define INIT_HASH_BITS 12

struct dictinit_state_s {
  uint32_t nnz;
  uint32_t total_dupes;
  DictionaryChunk chunk;
  volatile uint32_t scratch_red[32];
  uint16_t dict[MAX_SHORT_DICT_ENTRIES];
  union {
    uint16_t u16[1 << (INIT_HASH_BITS)];
    uint32_t u32[1 << (INIT_HASH_BITS - 1)];
  } map;
};

/**
 * @brief Return a 12-bit hash from a byte sequence
 */
static inline __device__ uint32_t nvstr_init_hash(const uint8_t *ptr, uint32_t len)
{
  if (len != 0) {
    return (ptr[0] + (ptr[len - 1] << 5) + (len << 10)) & ((1 << INIT_HASH_BITS) - 1);
  } else {
    return 0;
  }
}

/**
 * @brief Fill dictionary with the indices of non-null rows
 *
 * @param[in,out] s dictionary builder state
 * @param[in] t thread id
 *
 **/
static __device__ void LoadNonNullIndices(volatile dictinit_state_s *s, int t)
{
  if (t == 0) { s->nnz = 0; }
  for (uint32_t i = 0; i < s->chunk.num_rows; i += 512) {
    const uint32_t *valid_map = s->chunk.valid_map_base;
    uint32_t is_valid, nz_map, nz_pos;
    if (t < 16) {
      if (!valid_map) {
        s->scratch_red[t] = 0xffffffffu;
      } else {
        uint32_t row = s->chunk.start_row + i + t * 32;
        uint32_t v   = (row < s->chunk.start_row + s->chunk.num_rows) ? valid_map[row >> 5] : 0;
        if (row & 0x1f) {
          uint32_t v1 =
            (row + 32 < s->chunk.start_row + s->chunk.num_rows) ? valid_map[(row >> 5) + 1] : 0;
          v = __funnelshift_r(v, v1, row & 0x1f);
        }
        s->scratch_red[t] = v;
      }
    }
    __syncthreads();
    is_valid = (i + t < s->chunk.num_rows) ? (s->scratch_red[t >> 5] >> (t & 0x1f)) & 1 : 0;
    nz_map   = ballot(is_valid);
    nz_pos   = s->nnz + __popc(nz_map & (0x7fffffffu >> (0x1fu - ((uint32_t)t & 0x1f))));
    if (!(t & 0x1f)) { s->scratch_red[16 + (t >> 5)] = __popc(nz_map); }
    __syncthreads();
    if (t < 32) {
      uint32_t nnz     = s->scratch_red[16 + (t & 0xf)];
      uint32_t nnz_pos = WarpReducePos16(nnz, t);
      if (t == 0xf) { s->nnz += nnz_pos; }
      if (t <= 0xf) { s->scratch_red[t] = nnz_pos - nnz; }
    }
    __syncthreads();
    if (is_valid) { s->dict[nz_pos + s->scratch_red[t >> 5]] = i + t; }
    __syncthreads();
  }
}

/**
 * @brief Gather all non-NULL string rows and compute total character data size
 *
 * @param[in] chunks DictionaryChunk device array [rowgroup][column]
 * @param[in] num_columns Number of columns
 *
 **/
// blockDim {512,1,1}
template <int block_size>
__global__ void __launch_bounds__(block_size, 2)
  gpuInitDictionaryIndices(DictionaryChunk *chunks, uint32_t num_columns)
{
  __shared__ __align__(16) dictinit_state_s state_g;
  __shared__ union {
    typename hipcub::WarpReduce<uint32_t>::TempStorage full[block_size / 32];
    typename hipcub::WarpReduce<uint32_t, 16>::TempStorage half[block_size / 32];
  } temp_storage;

  dictinit_state_s *const s = &state_g;
  uint32_t col_id           = blockIdx.x;
  uint32_t group_id         = blockIdx.y;
  const nvstrdesc_s *ck_data;
  uint32_t *dict_data;
  uint32_t nnz, start_row, dict_char_count;
  int t = threadIdx.x;

  if (t < sizeof(DictionaryChunk) / sizeof(uint32_t)) {
    ((volatile uint32_t *)&s->chunk)[t] =
      ((const uint32_t *)&chunks[group_id * num_columns + col_id])[t];
  }
  for (uint32_t i = 0; i < sizeof(s->map) / sizeof(uint32_t); i += block_size) {
    if (i + t < sizeof(s->map) / sizeof(uint32_t)) s->map.u32[i + t] = 0;
  }
  __syncthreads();
  // First, take care of NULLs, and count how many strings we have (TODO: bypass this step when
  // there are no nulls)
  LoadNonNullIndices(s, t);
  // Sum the lengths of all the strings
  if (t == 0) {
    s->chunk.string_char_count = 0;
    s->total_dupes             = 0;
  }
  nnz       = s->nnz;
  dict_data = s->chunk.dict_data;
  start_row = s->chunk.start_row;
  ck_data   = reinterpret_cast<const nvstrdesc_s *>(s->chunk.column_data_base) + start_row;
  for (uint32_t i = 0; i < nnz; i += block_size) {
    uint32_t ck_row = 0, len = 0, hash;
    const uint8_t *ptr = 0;
    if (i + t < nnz) {
      ck_row = s->dict[i + t];
      ptr    = reinterpret_cast<const uint8_t *>(ck_data[ck_row].ptr);
      len    = ck_data[ck_row].count;
      hash   = nvstr_init_hash(ptr, len);
    }
    len = hipcub::WarpReduce<uint32_t, 16>(temp_storage.half[threadIdx.x / 32]).Sum(len);
    if (!(t & 0xf)) { s->scratch_red[t >> 4] = len; }
    __syncthreads();
    if (t < 32) {
      len = hipcub::WarpReduce<uint32_t>(temp_storage.full[threadIdx.x / 32]).Sum(s->scratch_red[t]);
      if (t == 0) s->chunk.string_char_count += len;
    }
    if (i + t < nnz) {
      atomicAdd(&s->map.u32[hash >> 1], 1 << ((hash & 1) ? 16 : 0));
      dict_data[i + t] = start_row + ck_row;
    }
    __syncthreads();
  }
  // Reorder the 16-bit local indices according to the hash value of the strings
#if (INIT_HASH_BITS != 12)
#error "Hardcoded for INIT_HASH_BITS=12"
#endif
  {
    // Cumulative sum of hash map counts
    uint32_t count01 = s->map.u32[t * 4 + 0];
    uint32_t count23 = s->map.u32[t * 4 + 1];
    uint32_t count45 = s->map.u32[t * 4 + 2];
    uint32_t count67 = s->map.u32[t * 4 + 3];
    uint32_t sum01   = count01 + (count01 << 16);
    uint32_t sum23   = count23 + (count23 << 16);
    uint32_t sum45   = count45 + (count45 << 16);
    uint32_t sum67   = count67 + (count67 << 16);
    uint32_t sum_w, tmp;
    sum23 += (sum01 >> 16) * 0x10001;
    sum45 += (sum23 >> 16) * 0x10001;
    sum67 += (sum45 >> 16) * 0x10001;
    sum_w = sum67 >> 16;
    sum_w = WarpReducePos16(sum_w, t);
    if ((t & 0xf) == 0xf) { s->scratch_red[t >> 4] = sum_w; }
    __syncthreads();
    if (t < 32) {
      uint32_t sum_b    = WarpReducePos32(s->scratch_red[t], t);
      s->scratch_red[t] = sum_b;
    }
    __syncthreads();
    tmp                   = (t >= 16) ? s->scratch_red[(t >> 4) - 1] : 0;
    sum_w                 = (sum_w - (sum67 >> 16) + tmp) * 0x10001;
    s->map.u32[t * 4 + 0] = sum_w + sum01 - count01;
    s->map.u32[t * 4 + 1] = sum_w + sum23 - count23;
    s->map.u32[t * 4 + 2] = sum_w + sum45 - count45;
    s->map.u32[t * 4 + 3] = sum_w + sum67 - count67;
    __syncthreads();
  }
  // Put the indices back in hash order
  for (uint32_t i = 0; i < nnz; i += block_size) {
    uint32_t ck_row = 0, pos = 0, hash = 0, pos_old, pos_new, sh, colliding_row;
    bool collision;
    if (i + t < nnz) {
      const uint8_t *ptr;
      uint32_t len;
      ck_row  = dict_data[i + t] - start_row;
      ptr     = reinterpret_cast<const uint8_t *>(ck_data[ck_row].ptr);
      len     = (uint32_t)ck_data[ck_row].count;
      hash    = nvstr_init_hash(ptr, len);
      sh      = (hash & 1) ? 16 : 0;
      pos_old = s->map.u16[hash];
    }
    // The isolation of the atomicAdd, along with pos_old/pos_new is to guarantee deterministic
    // behavior for the first row in the hash map that will be used for early duplicate detection
    // The lack of 16-bit atomicMin makes this a bit messy...
    __syncthreads();
    if (i + t < nnz) {
      pos          = (atomicAdd(&s->map.u32[hash >> 1], 1 << sh) >> sh) & 0xffff;
      s->dict[pos] = ck_row;
    }
    __syncthreads();
    collision = false;
    if (i + t < nnz) {
      pos_new   = s->map.u16[hash];
      collision = (pos != pos_old && pos_new > pos_old + 1);
      if (collision) { colliding_row = s->dict[pos_old]; }
    }
    __syncthreads();
    // evens
    if (collision && !(pos_old & 1)) {
      uint32_t *dict32 = reinterpret_cast<uint32_t *>(&s->dict[pos_old]);
      atomicMin(dict32, (dict32[0] & 0xffff0000) | ck_row);
    }
    __syncthreads();
    // odds
    if (collision && (pos_old & 1)) {
      uint32_t *dict32 = reinterpret_cast<uint32_t *>(&s->dict[pos_old - 1]);
      atomicMin(dict32, (dict32[0] & 0x0000ffff) | (ck_row << 16));
    }
    __syncthreads();
    // Resolve collision
    if (collision && ck_row == s->dict[pos_old]) { s->dict[pos] = colliding_row; }
  }
  __syncthreads();
  // Now that the strings are ordered by hash, compare every string with the first entry in the hash
  // map, the position of the first string can be inferred from the hash map counts
  dict_char_count = 0;
  for (uint32_t i = 0; i < nnz; i += block_size) {
    uint32_t ck_row = 0, ck_row_ref = 0, is_dupe = 0, dupe_mask, dupes_before;
    if (i + t < nnz) {
      const char *str1, *str2;
      uint32_t len1, len2, hash;
      ck_row     = s->dict[i + t];
      str1       = ck_data[ck_row].ptr;
      len1       = (uint32_t)ck_data[ck_row].count;
      hash       = nvstr_init_hash(reinterpret_cast<const uint8_t *>(str1), len1);
      ck_row_ref = s->dict[(hash > 0) ? s->map.u16[hash - 1] : 0];
      if (ck_row_ref != ck_row) {
        str2    = ck_data[ck_row_ref].ptr;
        len2    = (uint32_t)ck_data[ck_row_ref].count;
        is_dupe = nvstr_is_equal(str1, len1, str2, len2);
        dict_char_count += (is_dupe) ? 0 : len1;
      }
    }
    dupe_mask    = ballot(is_dupe);
    dupes_before = s->total_dupes + __popc(dupe_mask & ((2 << (t & 0x1f)) - 1));
    if (!(t & 0x1f)) { s->scratch_red[t >> 5] = __popc(dupe_mask); }
    __syncthreads();
    if (t < 32) {
      uint32_t warp_dupes = (t < 16) ? s->scratch_red[t] : 0;
      uint32_t warp_pos   = WarpReducePos16(warp_dupes, t);
      if (t == 0xf) { s->total_dupes += warp_pos; }
      if (t < 16) { s->scratch_red[t] = warp_pos - warp_dupes; }
    }
    __syncthreads();
    if (i + t < nnz) {
      if (!is_dupe) {
        dupes_before += s->scratch_red[t >> 5];
        dict_data[i + t - dupes_before] = ck_row + start_row;
      } else {
        s->chunk.dict_index[ck_row + start_row] = (ck_row_ref + start_row) | (1u << 31);
      }
    }
  }
  dict_char_count =
    hipcub::WarpReduce<uint32_t>(temp_storage.full[threadIdx.x / 32]).Sum(dict_char_count);
  if (!(t & 0x1f)) { s->scratch_red[t >> 5] = dict_char_count; }
  __syncthreads();
  if (t < 32) {
    dict_char_count = hipcub::WarpReduce<uint32_t, 16>(temp_storage.half[threadIdx.x / 32])
                        .Sum((t < 16) ? s->scratch_red[t] : 0);
  }
  if (!t) {
    chunks[group_id * num_columns + col_id].num_strings       = nnz;
    chunks[group_id * num_columns + col_id].string_char_count = s->chunk.string_char_count;
    chunks[group_id * num_columns + col_id].num_dict_strings  = nnz - s->total_dupes;
    chunks[group_id * num_columns + col_id].dict_char_count   = dict_char_count;
  }
}

/**
 * @brief In-place concatenate dictionary data for all chunks in each stripe
 *
 * @param[in] stripes StripeDictionary device array [stripe][column]
 * @param[in] chunks DictionaryChunk device array [rowgroup][column]
 * @param[in] num_columns Number of columns
 *
 **/
// blockDim {1024,1,1}
extern "C" __global__ void __launch_bounds__(1024)
  gpuCompactChunkDictionaries(StripeDictionary *stripes,
                              DictionaryChunk const *chunks,
                              uint32_t num_columns)
{
  __shared__ __align__(16) StripeDictionary stripe_g;
  __shared__ __align__(16) DictionaryChunk chunk_g;
  __shared__ const uint32_t *volatile ck_curptr_g;
  __shared__ uint32_t volatile ck_curlen_g;

  uint32_t col_id    = blockIdx.x;
  uint32_t stripe_id = blockIdx.y;
  uint32_t chunk_len;
  int t = threadIdx.x;
  const uint32_t *src;
  uint32_t *dst;

  if (t < sizeof(StripeDictionary) / sizeof(uint32_t)) {
    ((volatile uint32_t *)&stripe_g)[t] =
      ((const uint32_t *)&stripes[stripe_id * num_columns + col_id])[t];
  }
  __syncthreads();
  if (!stripe_g.dict_data) { return; }
  if (t < sizeof(DictionaryChunk) / sizeof(uint32_t)) {
    ((volatile uint32_t *)&chunk_g)[t] =
      ((const uint32_t *)&chunks[stripe_g.start_chunk * num_columns + col_id])[t];
  }
  __syncthreads();
  dst = stripe_g.dict_data + chunk_g.num_dict_strings;
  for (uint32_t g = 1; g < stripe_g.num_chunks; g++) {
    if (!t) {
      src         = chunks[(stripe_g.start_chunk + g) * num_columns + col_id].dict_data;
      chunk_len   = chunks[(stripe_g.start_chunk + g) * num_columns + col_id].num_dict_strings;
      ck_curptr_g = src;
      ck_curlen_g = chunk_len;
    }
    __syncthreads();
    src       = ck_curptr_g;
    chunk_len = ck_curlen_g;
    if (src != dst) {
      for (uint32_t i = 0; i < chunk_len; i += 1024) {
        uint32_t idx = (i + t < chunk_len) ? src[i + t] : 0;
        __syncthreads();
        if (i + t < chunk_len) dst[i + t] = idx;
      }
    }
    dst += chunk_len;
    __syncthreads();
  }
}

struct build_state_s {
  uint32_t total_dupes;
  StripeDictionary stripe;
  volatile uint32_t scratch_red[32];
};

/**
 * @brief Eliminate duplicates in-place and generate column dictionary index
 *
 * @param[in] stripes StripeDictionary device array [stripe][column]
 * @param[in] num_columns Number of string columns
 *
 **/
// NOTE: Prone to poor utilization on small datasets due to 1 block per dictionary
// blockDim {1024,1,1}
template <int block_size>
__global__ void __launch_bounds__(block_size)
  gpuBuildStripeDictionaries(StripeDictionary *stripes, uint32_t num_columns)
{
  __shared__ __align__(16) build_state_s state_g;
  using warp_reduce = hipcub::WarpReduce<uint32_t>;
  __shared__ typename warp_reduce::TempStorage temp_storage[block_size / 32];

  volatile build_state_s *const s = &state_g;
  uint32_t col_id                 = blockIdx.x;
  uint32_t stripe_id              = blockIdx.y;
  uint32_t num_strings;
  uint32_t *dict_data, *dict_index;
  uint32_t dict_char_count;
  const nvstrdesc_s *str_data;
  int t = threadIdx.x;

  if (t < sizeof(StripeDictionary) / sizeof(uint32_t)) {
    ((volatile uint32_t *)&s->stripe)[t] =
      ((const uint32_t *)&stripes[stripe_id * num_columns + col_id])[t];
  }
  if (t == 31 * 32) { s->total_dupes = 0; }
  __syncthreads();
  num_strings = s->stripe.num_strings;
  dict_data   = s->stripe.dict_data;
  if (!dict_data) return;
  dict_index      = s->stripe.dict_index;
  str_data        = reinterpret_cast<const nvstrdesc_s *>(s->stripe.column_data_base);
  dict_char_count = 0;
  for (uint32_t i = 0; i < num_strings; i += block_size) {
    uint32_t cur = (i + t < num_strings) ? dict_data[i + t] : 0;
    uint32_t dupe_mask, dupes_before, cur_len = 0;
    const char *cur_ptr;
    bool is_dupe = false;
    if (i + t < num_strings) {
      cur_ptr = str_data[cur].ptr;
      cur_len = str_data[cur].count;
    }
    if (i + t != 0 && i + t < num_strings) {
      uint32_t prev = dict_data[i + t - 1];
      is_dupe       = nvstr_is_equal(cur_ptr, cur_len, str_data[prev].ptr, str_data[prev].count);
    }
    dict_char_count += (is_dupe) ? 0 : cur_len;
    dupe_mask    = ballot(is_dupe);
    dupes_before = s->total_dupes + __popc(dupe_mask & ((2 << (t & 0x1f)) - 1));
    if (!(t & 0x1f)) { s->scratch_red[t >> 5] = __popc(dupe_mask); }
    __syncthreads();
    if (t < 32) {
      uint32_t warp_dupes = s->scratch_red[t];
      uint32_t warp_pos   = WarpReducePos32(warp_dupes, t);
      if (t == 0x1f) { s->total_dupes += warp_pos; }
      s->scratch_red[t] = warp_pos - warp_dupes;
    }
    __syncthreads();
    if (i + t < num_strings) {
      dupes_before += s->scratch_red[t >> 5];
      dict_index[cur] = i + t - dupes_before;
      if (!is_dupe && dupes_before != 0) { dict_data[i + t - dupes_before] = cur; }
    }
    __syncthreads();
  }
  dict_char_count = warp_reduce(temp_storage[threadIdx.x / 32]).Sum(dict_char_count);
  if (!(t & 0x1f)) { s->scratch_red[t >> 5] = dict_char_count; }
  __syncthreads();
  if (t < 32) {
    dict_char_count = warp_reduce(temp_storage[threadIdx.x / 32]).Sum(s->scratch_red[t]);
  }
  if (t == 0) {
    stripes[stripe_id * num_columns + col_id].num_strings     = num_strings - s->total_dupes;
    stripes[stripe_id * num_columns + col_id].dict_char_count = dict_char_count;
  }
}

/**
 * @brief Launches kernel for initializing dictionary chunks
 *
 * @param[in] chunks DictionaryChunk device array [rowgroup][column]
 * @param[in] num_columns Number of columns
 * @param[in] num_rowgroups Number of row groups
 * @param[in] stream CUDA stream to use, default 0
 *
 * @return hipSuccess if successful, a CUDA error code otherwise
 **/
hipError_t InitDictionaryIndices(DictionaryChunk *chunks,
                                  uint32_t num_columns,
                                  uint32_t num_rowgroups,
                                  hipStream_t stream)
{
  dim3 dim_block(512, 1);  // 512 threads per chunk
  dim3 dim_grid(num_columns, num_rowgroups);
  gpuInitDictionaryIndices<512><<<dim_grid, dim_block, 0, stream>>>(chunks, num_columns);
  return hipSuccess;
}

/**
 * @brief Launches kernel for building stripe dictionaries
 *
 * @param[in] stripes StripeDictionary device array [stripe][column]
 * @param[in] stripes_host StripeDictionary host array [stripe][column]
 * @param[in] chunks DictionaryChunk device array [rowgroup][column]
 * @param[in] num_stripes Number of stripes
 * @param[in] num_rowgroups Number of row groups
 * @param[in] num_columns Number of columns
 * @param[in] stream CUDA stream to use, default 0
 *
 * @return hipSuccess if successful, a CUDA error code otherwise
 **/
hipError_t BuildStripeDictionaries(StripeDictionary *stripes,
                                    StripeDictionary *stripes_host,
                                    DictionaryChunk const *chunks,
                                    uint32_t num_stripes,
                                    uint32_t num_rowgroups,
                                    uint32_t num_columns,
                                    hipStream_t stream)
{
  dim3 dim_block(1024, 1);  // 1024 threads per chunk
  dim3 dim_grid_build(num_columns, num_stripes);
  gpuCompactChunkDictionaries<<<dim_grid_build, dim_block, 0, stream>>>(
    stripes, chunks, num_columns);
  for (uint32_t i = 0; i < num_stripes * num_columns; i++) {
    if (stripes_host[i].dict_data != nullptr) {
      thrust::device_ptr<uint32_t> p = thrust::device_pointer_cast(stripes_host[i].dict_data);
      const nvstrdesc_s *str_data =
        reinterpret_cast<const nvstrdesc_s *>(stripes_host[i].column_data_base);
      // NOTE: Requires the --expt-extended-lambda nvcc flag
      thrust::sort(rmm::exec_policy(stream)->on(stream),
                   p,
                   p + stripes_host[i].num_strings,
                   [str_data] __device__(const uint32_t &lhs, const uint32_t &rhs) {
                     return nvstr_is_lesser(str_data[lhs].ptr,
                                            (uint32_t)str_data[lhs].count,
                                            str_data[rhs].ptr,
                                            (uint32_t)str_data[rhs].count);
                   });
    }
  }
  gpuBuildStripeDictionaries<1024><<<dim_grid_build, dim_block, 0, stream>>>(stripes, num_columns);
  return hipSuccess;
}

}  // namespace gpu
}  // namespace orc
}  // namespace io
}  // namespace cudf
