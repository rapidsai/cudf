#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019-2024, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "io/comp/gpuinflate.hpp"
#include "io/comp/nvcomp_adapter.hpp"
#include "io/utilities/config_utils.hpp"
#include "reader_impl.hpp"
#include "reader_impl_chunking.hpp"
#include "reader_impl_helpers.hpp"

#include <cudf/detail/timezone.hpp>
#include <cudf/detail/utilities/integer_utils.hpp>
#include <cudf/detail/utilities/logger.hpp>
#include <cudf/detail/utilities/vector_factories.hpp>
#include <cudf/table/table.hpp>
#include <cudf/utilities/bit.hpp>
#include <cudf/utilities/error.hpp>

#include <rmm/cuda_stream_view.hpp>
#include <rmm/device_buffer.hpp>
#include <rmm/device_scalar.hpp>
#include <rmm/device_uvector.hpp>
#include <rmm/exec_policy.hpp>
#include <rmm/resource_ref.hpp>

#include <thrust/copy.h>
#include <thrust/fill.h>
#include <thrust/for_each.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/pair.h>
#include <thrust/scan.h>
#include <thrust/transform.h>

#include <algorithm>
#include <iterator>

namespace cudf::io::orc::detail {

namespace {

/**
 * @brief Struct that maps ORC streams to columns
 */
struct orc_stream_info {
  explicit orc_stream_info(uint64_t offset_,
                           std::size_t dst_pos_,
                           uint32_t length_,
                           uint32_t stripe_idx_)
    : offset(offset_), dst_pos(dst_pos_), length(length_), stripe_idx(stripe_idx_)
  {
  }
  uint64_t offset;      // offset in file
  std::size_t dst_pos;  // offset in memory relative to start of compressed stripe data
  std::size_t length;   // length in file
  uint32_t stripe_idx;  // stripe index
};

/**
 * @brief Function that populates column descriptors stream/chunk
 */
std::size_t gather_stream_info(std::size_t stripe_index,
                               orc::StripeInformation const* stripeinfo,
                               orc::StripeFooter const* stripefooter,
                               host_span<int const> orc2gdf,
                               host_span<orc::SchemaType const> types,
                               bool use_index,
                               bool apply_struct_map,
                               int64_t* num_dictionary_entries,
                               std::vector<orc_stream_info>& stream_info,
                               cudf::detail::hostdevice_2dvector<gpu::ColumnDesc>& chunks)
{
  uint64_t src_offset = 0;
  uint64_t dst_offset = 0;

  auto const get_stream_index_type = [](orc::StreamKind kind) {
    switch (kind) {
      case orc::DATA: return gpu::CI_DATA;
      case orc::LENGTH:
      case orc::SECONDARY: return gpu::CI_DATA2;
      case orc::DICTIONARY_DATA: return gpu::CI_DICTIONARY;
      case orc::PRESENT: return gpu::CI_PRESENT;
      case orc::ROW_INDEX: return gpu::CI_INDEX;
      default:
        // Skip this stream as it's not strictly required
        return gpu::CI_NUM_STREAMS;
    }
  };

  for (auto const& stream : stripefooter->streams) {
    if (!stream.column_id || *stream.column_id >= orc2gdf.size()) {
      // Ignore reading this stream from source.
      cudf::logger().warn("Unexpected stream in the input ORC source. The stream will be ignored.");
      src_offset += stream.length;
      continue;
    }

    auto const column_id = *stream.column_id;
    auto col             = orc2gdf[column_id];

    if (col == -1 and apply_struct_map) {
      // A struct-type column has no data itself, but rather child columns
      // for each of its fields. There is only a PRESENT stream, which
      // needs to be included for the reader.
      auto const schema_type = types[column_id];
      if (not schema_type.subtypes.empty()) {
        if (schema_type.kind == orc::STRUCT && stream.kind == orc::PRESENT) {
          for (auto const& idx : schema_type.subtypes) {
            auto child_idx = (idx < orc2gdf.size()) ? orc2gdf[idx] : -1;
            if (child_idx >= 0) {
              col                             = child_idx;
              auto& chunk                     = chunks[stripe_index][col];
              chunk.strm_id[gpu::CI_PRESENT]  = stream_info.size();
              chunk.strm_len[gpu::CI_PRESENT] = stream.length;
            }
          }
        }
      }
    } else if (col != -1) {
      if (src_offset >= stripeinfo->indexLength || use_index) {
        auto& chunk           = chunks[stripe_index][col];
        auto const index_type = get_stream_index_type(stream.kind);
        if (index_type < gpu::CI_NUM_STREAMS) {
          chunk.strm_id[index_type]  = stream_info.size();
          chunk.strm_len[index_type] = stream.length;
          // NOTE: skip_count field is temporarily used to track the presence of index streams
          chunk.skip_count |= 1 << index_type;

          if (index_type == gpu::CI_DICTIONARY) {
            chunk.dictionary_start = *num_dictionary_entries;
            chunk.dict_len         = stripefooter->columns[column_id].dictionarySize;
            *num_dictionary_entries += stripefooter->columns[column_id].dictionarySize;
          }
        }
      }
      stream_info.emplace_back(
        stripeinfo->offset + src_offset, dst_offset, stream.length, stripe_index);
      dst_offset += stream.length;
    }
    src_offset += stream.length;
  }

  return dst_offset;
}

/**
 * @brief Decompresses the stripe data, at stream granularity.
 *
 * @param decompressor Block decompressor
 * @param stripe_data List of source stripe column data
 * @param stream_info List of stream to column mappings
 * @param chunks Vector of list of column chunk descriptors
 * @param row_groups Vector of list of row index descriptors
 * @param num_stripes Number of stripes making up column chunks
 * @param row_index_stride Distance between each row index
 * @param use_base_stride Whether to use base stride obtained from meta or use the computed value
 * @param stream CUDA stream used for device memory operations and kernel launches
 * @return Device buffer to decompressed page data
 */
rmm::device_buffer decompress_stripe_data(
  OrcDecompressor const& decompressor,
  host_span<rmm::device_buffer const> stripe_data,
  host_span<orc_stream_info> stream_info,
  cudf::detail::hostdevice_2dvector<gpu::ColumnDesc>& chunks,
  cudf::detail::hostdevice_2dvector<gpu::RowGroup>& row_groups,
  size_type num_stripes,
  size_type row_index_stride,
  bool use_base_stride,
  rmm::cuda_stream_view stream)
{
  // Parse the columns' compressed info
  cudf::detail::hostdevice_vector<gpu::CompressedStreamInfo> compinfo(
    0, stream_info.size(), stream);
  for (auto const& info : stream_info) {
    compinfo.push_back(gpu::CompressedStreamInfo(
      static_cast<uint8_t const*>(stripe_data[info.stripe_idx].data()) + info.dst_pos,
      info.length));
  }
  compinfo.host_to_device_async(stream);

  gpu::ParseCompressedStripeData(compinfo.device_ptr(),
                                 compinfo.size(),
                                 decompressor.GetBlockSize(),
                                 decompressor.GetLog2MaxCompressionRatio(),
                                 stream);
  compinfo.device_to_host_sync(stream);

  // Count the exact number of compressed blocks
  std::size_t num_compressed_blocks   = 0;
  std::size_t num_uncompressed_blocks = 0;
  std::size_t total_decomp_size       = 0;
  for (std::size_t i = 0; i < compinfo.size(); ++i) {
    num_compressed_blocks += compinfo[i].num_compressed_blocks;
    num_uncompressed_blocks += compinfo[i].num_uncompressed_blocks;
    total_decomp_size += compinfo[i].max_uncompressed_size;
  }
  CUDF_EXPECTS(
    not((num_uncompressed_blocks + num_compressed_blocks > 0) and (total_decomp_size == 0)),
    "Inconsistent info on compression blocks");

  // Buffer needs to be padded.
  // Required by `gpuDecodeOrcColumnData`.
  rmm::device_buffer decomp_data(
    cudf::util::round_up_safe(total_decomp_size, BUFFER_PADDING_MULTIPLE), stream);
  if (decomp_data.is_empty()) { return decomp_data; }

  rmm::device_uvector<device_span<uint8_t const>> inflate_in(
    num_compressed_blocks + num_uncompressed_blocks, stream);
  rmm::device_uvector<device_span<uint8_t>> inflate_out(
    num_compressed_blocks + num_uncompressed_blocks, stream);
  rmm::device_uvector<compression_result> inflate_res(num_compressed_blocks, stream);
  thrust::fill(rmm::exec_policy(stream),
               inflate_res.begin(),
               inflate_res.end(),
               compression_result{0, compression_status::FAILURE});

  // Parse again to populate the decompression input/output buffers
  std::size_t decomp_offset      = 0;
  uint32_t max_uncomp_block_size = 0;
  uint32_t start_pos             = 0;
  auto start_pos_uncomp          = (uint32_t)num_compressed_blocks;
  for (std::size_t i = 0; i < compinfo.size(); ++i) {
    auto dst_base                 = static_cast<uint8_t*>(decomp_data.data());
    compinfo[i].uncompressed_data = dst_base + decomp_offset;
    compinfo[i].dec_in_ctl        = inflate_in.data() + start_pos;
    compinfo[i].dec_out_ctl       = inflate_out.data() + start_pos;
    compinfo[i].dec_res      = {inflate_res.data() + start_pos, compinfo[i].num_compressed_blocks};
    compinfo[i].copy_in_ctl  = inflate_in.data() + start_pos_uncomp;
    compinfo[i].copy_out_ctl = inflate_out.data() + start_pos_uncomp;

    stream_info[i].dst_pos = decomp_offset;
    decomp_offset += compinfo[i].max_uncompressed_size;
    start_pos += compinfo[i].num_compressed_blocks;
    start_pos_uncomp += compinfo[i].num_uncompressed_blocks;
    max_uncomp_block_size =
      std::max(max_uncomp_block_size, compinfo[i].max_uncompressed_block_size);
  }
  compinfo.host_to_device_async(stream);
  gpu::ParseCompressedStripeData(compinfo.device_ptr(),
                                 compinfo.size(),
                                 decompressor.GetBlockSize(),
                                 decompressor.GetLog2MaxCompressionRatio(),
                                 stream);

  // Value for checking whether we decompress successfully.
  // It doesn't need to be atomic as there is no race condition: we only write `true` if needed.
  cudf::detail::hostdevice_vector<bool> any_block_failure(1, stream);
  any_block_failure[0] = false;
  any_block_failure.host_to_device_async(stream);

  // Dispatch batches of blocks to decompress
  if (num_compressed_blocks > 0) {
    device_span<device_span<uint8_t const>> inflate_in_view{inflate_in.data(),
                                                            num_compressed_blocks};
    device_span<device_span<uint8_t>> inflate_out_view{inflate_out.data(), num_compressed_blocks};
    switch (decompressor.compression()) {
      case compression_type::ZLIB:
        if (nvcomp::is_decompression_disabled(nvcomp::compression_type::DEFLATE)) {
          gpuinflate(
            inflate_in_view, inflate_out_view, inflate_res, gzip_header_included::NO, stream);
        } else {
          nvcomp::batched_decompress(nvcomp::compression_type::DEFLATE,
                                     inflate_in_view,
                                     inflate_out_view,
                                     inflate_res,
                                     max_uncomp_block_size,
                                     total_decomp_size,
                                     stream);
        }
        break;
      case compression_type::SNAPPY:
        if (nvcomp::is_decompression_disabled(nvcomp::compression_type::SNAPPY)) {
          gpu_unsnap(inflate_in_view, inflate_out_view, inflate_res, stream);
        } else {
          nvcomp::batched_decompress(nvcomp::compression_type::SNAPPY,
                                     inflate_in_view,
                                     inflate_out_view,
                                     inflate_res,
                                     max_uncomp_block_size,
                                     total_decomp_size,
                                     stream);
        }
        break;
      case compression_type::ZSTD:
        if (auto const reason = nvcomp::is_decompression_disabled(nvcomp::compression_type::ZSTD);
            reason) {
          CUDF_FAIL("Decompression error: " + reason.value());
        }
        nvcomp::batched_decompress(nvcomp::compression_type::ZSTD,
                                   inflate_in_view,
                                   inflate_out_view,
                                   inflate_res,
                                   max_uncomp_block_size,
                                   total_decomp_size,
                                   stream);
        break;
      case compression_type::LZ4:
        if (auto const reason = nvcomp::is_decompression_disabled(nvcomp::compression_type::LZ4);
            reason) {
          CUDF_FAIL("Decompression error: " + reason.value());
        }
        nvcomp::batched_decompress(nvcomp::compression_type::LZ4,
                                   inflate_in_view,
                                   inflate_out_view,
                                   inflate_res,
                                   max_uncomp_block_size,
                                   total_decomp_size,
                                   stream);
        break;
      default: CUDF_FAIL("Unexpected decompression dispatch"); break;
    }

    // Check if any block has been failed to decompress.
    // Not using `thrust::any` or `thrust::count_if` to defer stream sync.
    thrust::for_each(
      rmm::exec_policy(stream),
      thrust::make_counting_iterator(std::size_t{0}),
      thrust::make_counting_iterator(inflate_res.size()),
      [results           = inflate_res.begin(),
       any_block_failure = any_block_failure.device_ptr()] __device__(auto const idx) {
        if (results[idx].status != compression_status::SUCCESS) { *any_block_failure = true; }
      });
  }

  if (num_uncompressed_blocks > 0) {
    device_span<device_span<uint8_t const>> copy_in_view{inflate_in.data() + num_compressed_blocks,
                                                         num_uncompressed_blocks};
    device_span<device_span<uint8_t>> copy_out_view{inflate_out.data() + num_compressed_blocks,
                                                    num_uncompressed_blocks};
    gpu_copy_uncompressed_blocks(copy_in_view, copy_out_view, stream);
  }

  // Copy without stream sync, thus need to wait for stream sync below to access.
  any_block_failure.device_to_host_async(stream);

  gpu::PostDecompressionReassemble(compinfo.device_ptr(), compinfo.size(), stream);
  compinfo.device_to_host_sync(stream);  // This also sync stream for `any_block_failure`.

  // We can check on host after stream synchronize
  CUDF_EXPECTS(not any_block_failure[0], "Error during decompression");

  size_type const num_columns = chunks.size().second;

  // Update the stream information with the updated uncompressed info
  // TBD: We could update the value from the information we already
  // have in stream_info[], but using the gpu results also updates
  // max_uncompressed_size to the actual uncompressed size, or zero if
  // decompression failed.
  for (size_type i = 0; i < num_stripes; ++i) {
    for (size_type j = 0; j < num_columns; ++j) {
      auto& chunk = chunks[i][j];
      for (int k = 0; k < gpu::CI_NUM_STREAMS; ++k) {
        if (chunk.strm_len[k] > 0 && chunk.strm_id[k] < compinfo.size()) {
          chunk.streams[k]  = compinfo[chunk.strm_id[k]].uncompressed_data;
          chunk.strm_len[k] = compinfo[chunk.strm_id[k]].max_uncompressed_size;
        }
      }
    }
  }

  if (row_groups.size().first) {
    chunks.host_to_device_async(stream);
    row_groups.host_to_device_async(stream);
    gpu::ParseRowGroupIndex(row_groups.base_device_ptr(),
                            compinfo.device_ptr(),
                            chunks.base_device_ptr(),
                            num_columns,
                            num_stripes,
                            row_index_stride,
                            use_base_stride,
                            stream);
  }

  return decomp_data;
}

/**
 * @brief Updates null mask of columns whose parent is a struct column.
 *
 * If struct column has null element, that row would be skipped while writing child column in ORC,
 * so we need to insert the missing null elements in child column. There is another behavior from
 * pyspark, where if the child column doesn't have any null elements, it will not have present
 * stream, so in that case parent null mask need to be copied to child column.
 *
 * @param chunks Vector of list of column chunk descriptors
 * @param out_buffers Output columns' device buffers
 * @param stream CUDA stream used for device memory operations and kernel launches.
 * @param mr Device memory resource to use for device memory allocation
 */
void update_null_mask(cudf::detail::hostdevice_2dvector<gpu::ColumnDesc>& chunks,
                      host_span<column_buffer> out_buffers,
                      rmm::cuda_stream_view stream,
                      rmm::device_async_resource_ref mr)
{
  auto const num_stripes = chunks.size().first;
  auto const num_columns = chunks.size().second;
  bool is_mask_updated   = false;

  for (std::size_t col_idx = 0; col_idx < num_columns; ++col_idx) {
    if (chunks[0][col_idx].parent_validity_info.valid_map_base != nullptr) {
      if (not is_mask_updated) {
        chunks.device_to_host_sync(stream);
        is_mask_updated = true;
      }

      auto parent_valid_map_base = chunks[0][col_idx].parent_validity_info.valid_map_base;
      auto child_valid_map_base  = out_buffers[col_idx].null_mask();
      auto child_mask_len =
        chunks[0][col_idx].column_num_rows - chunks[0][col_idx].parent_validity_info.null_count;
      auto parent_mask_len = chunks[0][col_idx].column_num_rows;

      if (child_valid_map_base != nullptr) {
        rmm::device_uvector<uint32_t> dst_idx(child_mask_len, stream);
        // Copy indexes at which the parent has valid value.
        thrust::copy_if(rmm::exec_policy(stream),
                        thrust::make_counting_iterator(0),
                        thrust::make_counting_iterator(0) + parent_mask_len,
                        dst_idx.begin(),
                        [parent_valid_map_base] __device__(auto idx) {
                          return bit_is_set(parent_valid_map_base, idx);
                        });

        auto merged_null_mask = cudf::detail::create_null_mask(
          parent_mask_len, mask_state::ALL_NULL, rmm::cuda_stream_view(stream), mr);
        auto merged_mask      = static_cast<bitmask_type*>(merged_null_mask.data());
        uint32_t* dst_idx_ptr = dst_idx.data();
        // Copy child valid bits from child column to valid indexes, this will merge both child
        // and parent null masks
        thrust::for_each(rmm::exec_policy(stream),
                         thrust::make_counting_iterator(0),
                         thrust::make_counting_iterator(0) + dst_idx.size(),
                         [child_valid_map_base, dst_idx_ptr, merged_mask] __device__(auto idx) {
                           if (bit_is_set(child_valid_map_base, idx)) {
                             cudf::set_bit(merged_mask, dst_idx_ptr[idx]);
                           };
                         });

        out_buffers[col_idx].set_null_mask(std::move(merged_null_mask));

      } else {
        // Since child column doesn't have a mask, copy parent null mask
        auto mask_size = bitmask_allocation_size_bytes(parent_mask_len);
        out_buffers[col_idx].set_null_mask(
          rmm::device_buffer(static_cast<void*>(parent_valid_map_base), mask_size, stream, mr));
      }
    }
  }

  if (is_mask_updated) {
    // Update chunks with pointers to column data which might have been changed.
    for (std::size_t stripe_idx = 0; stripe_idx < num_stripes; ++stripe_idx) {
      for (std::size_t col_idx = 0; col_idx < num_columns; ++col_idx) {
        auto& chunk          = chunks[stripe_idx][col_idx];
        chunk.valid_map_base = out_buffers[col_idx].null_mask();
      }
    }
    chunks.host_to_device_sync(stream);
  }
}

/**
 * @brief Converts the stripe column data and outputs to columns.
 *
 * @param num_dicts Number of dictionary entries required
 * @param skip_rows Number of rows to offset from start
 * @param row_index_stride Distance between each row index
 * @param level Current nesting level being processed
 * @param tz_table Local time to UTC conversion table
 * @param chunks Vector of list of column chunk descriptors
 * @param row_groups Vector of list of row index descriptors
 * @param out_buffers Output columns' device buffers
 * @param stream CUDA stream used for device memory operations and kernel launches
 * @param mr Device memory resource to use for device memory allocation
 */
void decode_stream_data(std::size_t num_dicts,
                        int64_t skip_rows,
                        size_type row_index_stride,
                        std::size_t level,
                        table_view const& tz_table,
                        cudf::detail::hostdevice_2dvector<gpu::ColumnDesc>& chunks,
                        cudf::detail::device_2dspan<gpu::RowGroup> row_groups,
                        std::vector<column_buffer>& out_buffers,
                        rmm::cuda_stream_view stream,
                        rmm::device_async_resource_ref mr)
{
  auto const num_stripes = chunks.size().first;
  auto const num_columns = chunks.size().second;
  thrust::counting_iterator<int> col_idx_it(0);
  thrust::counting_iterator<int> stripe_idx_it(0);

  // Update chunks with pointers to column data
  std::for_each(stripe_idx_it, stripe_idx_it + num_stripes, [&](auto stripe_idx) {
    std::for_each(col_idx_it, col_idx_it + num_columns, [&](auto col_idx) {
      auto& chunk            = chunks[stripe_idx][col_idx];
      chunk.column_data_base = out_buffers[col_idx].data();
      chunk.valid_map_base   = out_buffers[col_idx].null_mask();
    });
  });

  // Allocate global dictionary for deserializing
  rmm::device_uvector<gpu::DictionaryEntry> global_dict(num_dicts, stream);

  chunks.host_to_device_sync(stream);
  gpu::DecodeNullsAndStringDictionaries(
    chunks.base_device_ptr(), global_dict.data(), num_columns, num_stripes, skip_rows, stream);

  if (level > 0) {
    // Update nullmasks for children if parent was a struct and had null mask
    update_null_mask(chunks, out_buffers, stream, mr);
  }

  auto const tz_table_dptr = table_device_view::create(tz_table, stream);
  rmm::device_scalar<size_type> error_count(0, stream);
  // Update the null map for child columns
  gpu::DecodeOrcColumnData(chunks.base_device_ptr(),
                           global_dict.data(),
                           row_groups,
                           num_columns,
                           num_stripes,
                           skip_rows,
                           *tz_table_dptr,
                           row_groups.size().first,
                           row_index_stride,
                           level,
                           error_count.data(),
                           stream);
  chunks.device_to_host_async(stream);
  // `value` synchronizes
  auto const num_errors = error_count.value(stream);
  CUDF_EXPECTS(num_errors == 0, "ORC data decode failed");

  std::for_each(col_idx_it + 0, col_idx_it + num_columns, [&](auto col_idx) {
    out_buffers[col_idx].null_count() =
      std::accumulate(stripe_idx_it + 0,
                      stripe_idx_it + num_stripes,
                      0,
                      [&](auto null_count, auto const stripe_idx) {
                        return null_count + chunks[stripe_idx][col_idx].null_count;
                      });
  });
}

/**
 * @brief Compute the per-stripe prefix sum of null count, for each struct column in the current
 * layer.
 */
void scan_null_counts(cudf::detail::hostdevice_2dvector<gpu::ColumnDesc> const& chunks,
                      cudf::host_span<rmm::device_uvector<uint32_t>> prefix_sums,
                      rmm::cuda_stream_view stream)
{
  auto const num_stripes = chunks.size().first;
  if (num_stripes == 0) return;

  auto const num_columns = chunks.size().second;
  std::vector<thrust::pair<size_type, cudf::device_span<uint32_t>>> prefix_sums_to_update;
  for (auto col_idx = 0ul; col_idx < num_columns; ++col_idx) {
    // Null counts sums are only needed for children of struct columns
    if (chunks[0][col_idx].type_kind == STRUCT) {
      prefix_sums_to_update.emplace_back(col_idx, prefix_sums[col_idx]);
    }
  }
  auto const d_prefix_sums_to_update = cudf::detail::make_device_uvector_async(
    prefix_sums_to_update, stream, rmm::mr::get_current_device_resource());

  thrust::for_each(rmm::exec_policy(stream),
                   d_prefix_sums_to_update.begin(),
                   d_prefix_sums_to_update.end(),
                   [chunks = cudf::detail::device_2dspan<gpu::ColumnDesc const>{chunks}] __device__(
                     auto const& idx_psums) {
                     auto const col_idx = idx_psums.first;
                     auto const psums   = idx_psums.second;

                     thrust::transform(
                       thrust::seq,
                       thrust::make_counting_iterator(0),
                       thrust::make_counting_iterator(0) + psums.size(),
                       psums.begin(),
                       [&](auto stripe_idx) { return chunks[stripe_idx][col_idx].null_count; });

                     thrust::inclusive_scan(thrust::seq, psums.begin(), psums.end(), psums.begin());
                   });
  // `prefix_sums_to_update` goes out of scope, copy has to be done before we return
  stream.synchronize();
}

/**
 * @brief Aggregate child metadata from parent column chunks.
 */
void aggregate_child_meta(std::size_t level,
                          cudf::io::orc::detail::column_hierarchy const& selected_columns,
                          cudf::detail::host_2dspan<gpu::ColumnDesc> chunks,
                          cudf::detail::host_2dspan<gpu::RowGroup> row_groups,
                          host_span<orc_column_meta const> nested_cols,
                          host_span<column_buffer> out_buffers,
                          reader_column_meta& col_meta)
{
  auto const num_of_stripes         = chunks.size().first;
  auto const num_of_rowgroups       = row_groups.size().first;
  auto const num_child_cols         = selected_columns.levels[level + 1].size();
  auto const number_of_child_chunks = num_child_cols * num_of_stripes;
  auto& num_child_rows              = col_meta.num_child_rows;
  auto& parent_column_data          = col_meta.parent_column_data;

  // Reset the meta to store child column details.
  num_child_rows.resize(selected_columns.levels[level + 1].size());
  std::fill(num_child_rows.begin(), num_child_rows.end(), 0);
  parent_column_data.resize(number_of_child_chunks);
  col_meta.parent_column_index.resize(number_of_child_chunks);
  col_meta.child_start_row.resize(number_of_child_chunks);
  col_meta.num_child_rows_per_stripe.resize(number_of_child_chunks);
  col_meta.rwgrp_meta.resize(num_of_rowgroups * num_child_cols);

  auto child_start_row = cudf::detail::host_2dspan<int64_t>(
    col_meta.child_start_row.data(), num_of_stripes, num_child_cols);
  auto num_child_rows_per_stripe = cudf::detail::host_2dspan<int64_t>(
    col_meta.num_child_rows_per_stripe.data(), num_of_stripes, num_child_cols);
  auto rwgrp_meta = cudf::detail::host_2dspan<reader_column_meta::row_group_meta>(
    col_meta.rwgrp_meta.data(), num_of_rowgroups, num_child_cols);

  int index = 0;  // number of child column processed

  // For each parent column, update its child column meta for each stripe.
  std::for_each(nested_cols.begin(), nested_cols.end(), [&](auto const p_col) {
    auto const parent_col_idx = col_meta.orc_col_map[level][p_col.id];
    int64_t start_row         = 0;
    auto processed_row_groups = 0;

    for (std::size_t stripe_id = 0; stripe_id < num_of_stripes; stripe_id++) {
      // Aggregate num_rows and start_row from processed parent columns per row groups
      if (num_of_rowgroups) {
        auto stripe_num_row_groups = chunks[stripe_id][parent_col_idx].num_rowgroups;
        auto processed_child_rows  = 0;

        for (std::size_t rowgroup_id = 0; rowgroup_id < stripe_num_row_groups;
             rowgroup_id++, processed_row_groups++) {
          auto const child_rows = row_groups[processed_row_groups][parent_col_idx].num_child_rows;
          for (size_type id = 0; id < p_col.num_children; id++) {
            auto const child_col_idx                                  = index + id;
            rwgrp_meta[processed_row_groups][child_col_idx].start_row = processed_child_rows;
            rwgrp_meta[processed_row_groups][child_col_idx].num_rows  = child_rows;
          }
          processed_child_rows += child_rows;
        }
      }

      // Aggregate start row, number of rows per chunk and total number of rows in a column
      auto const child_rows = chunks[stripe_id][parent_col_idx].num_child_rows;
      for (size_type id = 0; id < p_col.num_children; id++) {
        auto const child_col_idx = index + id;

        num_child_rows[child_col_idx] += child_rows;
        num_child_rows_per_stripe[stripe_id][child_col_idx] = child_rows;
        // start row could be different for each column when there is nesting at each stripe level
        child_start_row[stripe_id][child_col_idx] = (stripe_id == 0) ? 0 : start_row;
      }
      start_row += child_rows;
    }

    // Parent column null mask and null count would be required for child column
    // to adjust its nullmask.
    auto type              = out_buffers[parent_col_idx].type.id();
    auto parent_null_count = static_cast<uint32_t>(out_buffers[parent_col_idx].null_count());
    auto parent_valid_map  = out_buffers[parent_col_idx].null_mask();
    auto num_rows          = out_buffers[parent_col_idx].size;

    for (size_type id = 0; id < p_col.num_children; id++) {
      auto const child_col_idx                    = index + id;
      col_meta.parent_column_index[child_col_idx] = parent_col_idx;
      if (type == type_id::STRUCT) {
        parent_column_data[child_col_idx] = {parent_valid_map, parent_null_count};
        // Number of rows in child will remain same as parent in case of struct column
        num_child_rows[child_col_idx] = num_rows;
      } else {
        parent_column_data[child_col_idx] = {nullptr, 0};
      }
    }
    index += p_col.num_children;
  });
}

/**
 * @brief struct to store buffer data and size of list buffer
 */
struct list_buffer_data {
  size_type* data;
  size_type size;
};

// Generates offsets for list buffer from number of elements in a row.
void generate_offsets_for_list(host_span<list_buffer_data> buff_data, rmm::cuda_stream_view stream)
{
  for (auto& list_data : buff_data) {
    thrust::exclusive_scan(rmm::exec_policy_nosync(stream),
                           list_data.data,
                           list_data.data + list_data.size,
                           list_data.data);
  }
}

}  // namespace

void reader::impl::prepare_data(int64_t skip_rows,
                                std::optional<size_type> const& num_rows_opt,
                                std::vector<std::vector<size_type>> const& stripes)
{
  // Selected columns at different levels of nesting are stored in different elements
  // of `selected_columns`; thus, size == 1 means no nested columns
  CUDF_EXPECTS(skip_rows == 0 or _selected_columns.num_levels() == 1,
               "skip_rows is not supported by nested columns");

  // There are no columns in the table
  if (_selected_columns.num_levels() == 0) { return; }

  _file_itm_data = std::make_unique<file_intermediate_data>();

  // Select only stripes required (aka row groups)
  std::tie(
    _file_itm_data->rows_to_skip, _file_itm_data->rows_to_read, _file_itm_data->selected_stripes) =
    _metadata.select_stripes(stripes, skip_rows, num_rows_opt, _stream);
  auto const rows_to_skip      = _file_itm_data->rows_to_skip;
  auto const rows_to_read      = _file_itm_data->rows_to_read;
  auto const& selected_stripes = _file_itm_data->selected_stripes;

  // If no rows or stripes to read, return empty columns
  if (rows_to_read == 0 || selected_stripes.empty()) { return; }

  // Set up table for converting timestamp columns from local to UTC time
  auto const tz_table = [&, &selected_stripes = selected_stripes] {
    auto const has_timestamp_column = std::any_of(
      _selected_columns.levels.cbegin(), _selected_columns.levels.cend(), [&](auto const& col_lvl) {
        return std::any_of(col_lvl.cbegin(), col_lvl.cend(), [&](auto const& col_meta) {
          return _metadata.get_col_type(col_meta.id).kind == TypeKind::TIMESTAMP;
        });
      });

    return has_timestamp_column
             ? cudf::detail::make_timezone_transition_table(
                 {}, selected_stripes[0].stripe_info[0].second->writerTimezone, _stream)
             : std::make_unique<cudf::table>();
  }();

  auto& lvl_stripe_data        = _file_itm_data->lvl_stripe_data;
  auto& null_count_prefix_sums = _file_itm_data->null_count_prefix_sums;
  lvl_stripe_data.resize(_selected_columns.num_levels());

  _out_buffers.resize(_selected_columns.num_levels());

  // Iterates through levels of nested columns, child column will be one level down
  // compared to parent column.
  auto& col_meta = *_col_meta;
  for (std::size_t level = 0; level < _selected_columns.num_levels(); ++level) {
    auto& columns_level = _selected_columns.levels[level];
    // Association between each ORC column and its cudf::column
    col_meta.orc_col_map.emplace_back(_metadata.get_num_cols(), -1);
    std::vector<orc_column_meta> nested_cols;

    // Get a list of column data types
    std::vector<data_type> column_types;
    for (auto& col : columns_level) {
      auto col_type = to_cudf_type(_metadata.get_col_type(col.id).kind,
                                   _use_np_dtypes,
                                   _timestamp_type.id(),
                                   to_cudf_decimal_type(_decimal128_columns, _metadata, col.id));
      CUDF_EXPECTS(col_type != type_id::EMPTY, "Unknown type");
      if (col_type == type_id::DECIMAL32 or col_type == type_id::DECIMAL64 or
          col_type == type_id::DECIMAL128) {
        // sign of the scale is changed since cuDF follows c++ libraries like CNL
        // which uses negative scaling, but liborc and other libraries
        // follow positive scaling.
        auto const scale =
          -static_cast<size_type>(_metadata.get_col_type(col.id).scale.value_or(0));
        column_types.emplace_back(col_type, scale);
      } else {
        column_types.emplace_back(col_type);
      }

      // Map each ORC column to its column
      col_meta.orc_col_map[level][col.id] = column_types.size() - 1;
      if (col_type == type_id::LIST or col_type == type_id::STRUCT) {
        nested_cols.emplace_back(col);
      }
    }

    // Get the total number of stripes across all input files.
    std::size_t total_num_stripes =
      std::accumulate(selected_stripes.begin(),
                      selected_stripes.end(),
                      0,
                      [](std::size_t sum, auto& stripe_source_mapping) {
                        return sum + stripe_source_mapping.stripe_info.size();
                      });
    auto const num_columns = columns_level.size();
    cudf::detail::hostdevice_2dvector<gpu::ColumnDesc> chunks(
      total_num_stripes, num_columns, _stream);
    memset(chunks.base_host_ptr(), 0, chunks.size_bytes());

    const bool use_index =
      _use_index &&
      // Do stripes have row group index
      _metadata.is_row_grp_idx_present() &&
      // Only use if we don't have much work with complete columns & stripes
      // TODO: Consider nrows, gpu, and tune the threshold
      (rows_to_read > _metadata.get_row_index_stride() && !(_metadata.get_row_index_stride() & 7) &&
       _metadata.get_row_index_stride() != 0 && num_columns * total_num_stripes < 8 * 128) &&
      // Only use if first row is aligned to a stripe boundary
      // TODO: Fix logic to handle unaligned rows
      (rows_to_skip == 0);

    // Logically view streams as columns
    std::vector<orc_stream_info> stream_info;

    null_count_prefix_sums.emplace_back();
    null_count_prefix_sums.back().reserve(_selected_columns.levels[level].size());
    std::generate_n(std::back_inserter(null_count_prefix_sums.back()),
                    _selected_columns.levels[level].size(),
                    [&]() {
                      return cudf::detail::make_zeroed_device_uvector_async<uint32_t>(
                        total_num_stripes, _stream, rmm::mr::get_current_device_resource());
                    });

    // Tracker for eventually deallocating compressed and uncompressed data
    auto& stripe_data = lvl_stripe_data[level];

    int64_t stripe_start_row = 0;
    int64_t num_dict_entries = 0;
    int64_t num_rowgroups    = 0;
    size_type stripe_idx     = 0;

    std::vector<std::pair<std::future<std::size_t>, std::size_t>> read_tasks;
    for (auto const& stripe_source_mapping : selected_stripes) {
      // Iterate through the source files selected stripes
      for (auto const& stripe : stripe_source_mapping.stripe_info) {
        auto const stripe_info   = stripe.first;
        auto const stripe_footer = stripe.second;

        auto stream_count          = stream_info.size();
        auto const total_data_size = gather_stream_info(stripe_idx,
                                                        stripe_info,
                                                        stripe_footer,
                                                        col_meta.orc_col_map[level],
                                                        _metadata.get_types(),
                                                        use_index,
                                                        level == 0,
                                                        &num_dict_entries,
                                                        stream_info,
                                                        chunks);

        auto const is_stripe_data_empty = total_data_size == 0;
        CUDF_EXPECTS(not is_stripe_data_empty or stripe_info->indexLength == 0,
                     "Invalid index rowgroup stream data");

        // Buffer needs to be padded.
        // Required by `copy_uncompressed_kernel`.
        stripe_data.emplace_back(
          cudf::util::round_up_safe(total_data_size, BUFFER_PADDING_MULTIPLE), _stream);
        auto dst_base = static_cast<uint8_t*>(stripe_data.back().data());

        // Coalesce consecutive streams into one read
        while (not is_stripe_data_empty and stream_count < stream_info.size()) {
          auto const d_dst  = dst_base + stream_info[stream_count].dst_pos;
          auto const offset = stream_info[stream_count].offset;
          auto len          = stream_info[stream_count].length;
          stream_count++;

          while (stream_count < stream_info.size() &&
                 stream_info[stream_count].offset == offset + len) {
            len += stream_info[stream_count].length;
            stream_count++;
          }
          if (_metadata.per_file_metadata[stripe_source_mapping.source_idx]
                .source->is_device_read_preferred(len)) {
            read_tasks.push_back(
              std::pair(_metadata.per_file_metadata[stripe_source_mapping.source_idx]
                          .source->device_read_async(offset, len, d_dst, _stream),
                        len));

          } else {
            auto const buffer =
              _metadata.per_file_metadata[stripe_source_mapping.source_idx].source->host_read(
                offset, len);
            CUDF_EXPECTS(buffer->size() == len, "Unexpected discrepancy in bytes read.");
            CUDF_CUDA_TRY(
              hipMemcpyAsync(d_dst, buffer->data(), len, hipMemcpyDefault, _stream.value()));
            _stream.synchronize();
          }
        }

        auto const num_rows_per_stripe = stripe_info->numberOfRows;
        auto const rowgroup_id         = num_rowgroups;
        auto stripe_num_rowgroups      = 0;
        if (use_index) {
          stripe_num_rowgroups = (num_rows_per_stripe + _metadata.get_row_index_stride() - 1) /
                                 _metadata.get_row_index_stride();
        }
        // Update chunks to reference streams pointers
        for (std::size_t col_idx = 0; col_idx < num_columns; col_idx++) {
          auto& chunk = chunks[stripe_idx][col_idx];
          // start row, number of rows in a each stripe and total number of rows
          // may change in lower levels of nesting
          chunk.start_row = (level == 0)
                              ? stripe_start_row
                              : col_meta.child_start_row[stripe_idx * num_columns + col_idx];
          chunk.num_rows =
            (level == 0) ? stripe_info->numberOfRows
                         : col_meta.num_child_rows_per_stripe[stripe_idx * num_columns + col_idx];
          chunk.column_num_rows = (level == 0) ? rows_to_read : col_meta.num_child_rows[col_idx];
          chunk.parent_validity_info =
            (level == 0) ? column_validity_info{} : col_meta.parent_column_data[col_idx];
          chunk.parent_null_count_prefix_sums =
            (level == 0)
              ? nullptr
              : null_count_prefix_sums[level - 1][col_meta.parent_column_index[col_idx]].data();
          chunk.encoding_kind = stripe_footer->columns[columns_level[col_idx].id].kind;
          chunk.type_kind     = _metadata.per_file_metadata[stripe_source_mapping.source_idx]
                              .ff.types[columns_level[col_idx].id]
                              .kind;
          // num_child_rows for a struct column will be same, for other nested types it will be
          // calculated.
          chunk.num_child_rows = (chunk.type_kind != orc::STRUCT) ? 0 : chunk.num_rows;
          chunk.dtype_id       = column_types[col_idx].id();
          chunk.decimal_scale  = _metadata.per_file_metadata[stripe_source_mapping.source_idx]
                                  .ff.types[columns_level[col_idx].id]
                                  .scale.value_or(0);

          chunk.rowgroup_id   = rowgroup_id;
          chunk.dtype_len     = (column_types[col_idx].id() == type_id::STRING)
                                  ? sizeof(string_index_pair)
                                : ((column_types[col_idx].id() == type_id::LIST) or
                               (column_types[col_idx].id() == type_id::STRUCT))
                                  ? sizeof(size_type)
                                  : cudf::size_of(column_types[col_idx]);
          chunk.num_rowgroups = stripe_num_rowgroups;
          if (chunk.type_kind == orc::TIMESTAMP) { chunk.timestamp_type_id = _timestamp_type.id(); }
          if (not is_stripe_data_empty) {
            for (int k = 0; k < gpu::CI_NUM_STREAMS; k++) {
              chunk.streams[k] = dst_base + stream_info[chunk.strm_id[k]].dst_pos;
            }
          }
        }
        stripe_start_row += num_rows_per_stripe;
        num_rowgroups += stripe_num_rowgroups;

        stripe_idx++;
      }
    }
    for (auto& task : read_tasks) {
      CUDF_EXPECTS(task.first.get() == task.second, "Unexpected discrepancy in bytes read.");
    }

    if (stripe_data.empty()) { continue; }

    // Process dataset chunk pages into output columns
    auto row_groups =
      cudf::detail::hostdevice_2dvector<gpu::RowGroup>(num_rowgroups, num_columns, _stream);
    if (level > 0 and row_groups.size().first) {
      cudf::host_span<gpu::RowGroup> row_groups_span(row_groups.base_host_ptr(),
                                                     num_rowgroups * num_columns);
      auto& rw_grp_meta = col_meta.rwgrp_meta;

      // Update start row and num rows per row group
      std::transform(rw_grp_meta.begin(),
                     rw_grp_meta.end(),
                     row_groups_span.begin(),
                     rw_grp_meta.begin(),
                     [&](auto meta, auto& row_grp) {
                       row_grp.num_rows  = meta.num_rows;
                       row_grp.start_row = meta.start_row;
                       return meta;
                     });
    }
    // Setup row group descriptors if using indexes
    if (_metadata.per_file_metadata[0].ps.compression != orc::NONE) {
      auto decomp_data = decompress_stripe_data(*_metadata.per_file_metadata[0].decompressor,
                                                stripe_data,
                                                stream_info,
                                                chunks,
                                                row_groups,
                                                total_num_stripes,
                                                _metadata.get_row_index_stride(),
                                                level == 0,
                                                _stream);
      stripe_data.clear();
      stripe_data.push_back(std::move(decomp_data));
    } else {
      if (row_groups.size().first) {
        chunks.host_to_device_async(_stream);
        row_groups.host_to_device_async(_stream);
        row_groups.host_to_device_async(_stream);
        gpu::ParseRowGroupIndex(row_groups.base_device_ptr(),
                                nullptr,
                                chunks.base_device_ptr(),
                                num_columns,
                                total_num_stripes,
                                _metadata.get_row_index_stride(),
                                level == 0,
                                _stream);
      }
    }

    for (std::size_t i = 0; i < column_types.size(); ++i) {
      bool is_nullable = false;
      for (std::size_t j = 0; j < total_num_stripes; ++j) {
        if (chunks[j][i].strm_len[gpu::CI_PRESENT] != 0) {
          is_nullable = true;
          break;
        }
      }
      auto is_list_type = (column_types[i].id() == type_id::LIST);
      auto n_rows       = (level == 0) ? rows_to_read : col_meta.num_child_rows[i];
      // For list column, offset column will be always size + 1
      if (is_list_type) n_rows++;
      _out_buffers[level].emplace_back(column_types[i], n_rows, is_nullable, _stream, _mr);
    }

    decode_stream_data(num_dict_entries,
                       rows_to_skip,
                       _metadata.get_row_index_stride(),
                       level,
                       tz_table->view(),
                       chunks,
                       row_groups,
                       _out_buffers[level],
                       _stream,
                       _mr);

    if (nested_cols.size()) {
      // Extract information to process nested child columns
      scan_null_counts(chunks, null_count_prefix_sums[level], _stream);

      row_groups.device_to_host_sync(_stream);
      aggregate_child_meta(
        level, _selected_columns, chunks, row_groups, nested_cols, _out_buffers[level], col_meta);

      // ORC stores number of elements at each row, so we need to generate offsets from that
      std::vector<list_buffer_data> buff_data;
      std::for_each(
        _out_buffers[level].begin(), _out_buffers[level].end(), [&buff_data](auto& out_buffer) {
          if (out_buffer.type.id() == type_id::LIST) {
            auto data = static_cast<size_type*>(out_buffer.data());
            buff_data.emplace_back(list_buffer_data{data, out_buffer.size});
          }
        });

      if (not buff_data.empty()) { generate_offsets_for_list(buff_data, _stream); }
    }
  }  // end loop level
}

}  // namespace cudf::io::orc::detail
