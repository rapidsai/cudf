#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2020, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */
#include <io/utilities/block_utils.cuh>
#include "orc_common.h"
#include "orc_gpu.h"

namespace cudf {
namespace io {
namespace orc {
namespace gpu {
/**
 * @brief Initializes statistics groups
 *
 * @param[out] groups Statistics groups
 * @param[in] cols Column descriptors
 * @param[in] num_columns Number of columns
 * @param[in] num_rowgroups Number of rowgroups
 * @param[in] row_index_stride Rowgroup size in rows
 *
 **/
constexpr unsigned int init_threads_per_group = 32;
constexpr unsigned int init_groups_per_block  = 4;
constexpr unsigned int init_threads_per_block = init_threads_per_group * init_groups_per_block;

__global__ void __launch_bounds__(init_threads_per_block)
  gpu_init_statistics_groups(statistics_group *groups,
                             const stats_column_desc *cols,
                             uint32_t num_columns,
                             uint32_t num_rowgroups,
                             uint32_t row_index_stride)
{
  __shared__ __align__(4) statistics_group group_g[init_groups_per_block];
  uint32_t col_id         = blockIdx.y;
  uint32_t chunk_id       = (blockIdx.x * init_groups_per_block) + threadIdx.y;
  uint32_t t              = threadIdx.x;
  statistics_group *group = &group_g[threadIdx.y];
  if (chunk_id < num_rowgroups and t == 0) {
    uint32_t num_rows = cols[col_id].num_rows;
    group->col        = &cols[col_id];
    group->start_row  = chunk_id * row_index_stride;
    group->num_rows = min(num_rows - min(chunk_id * row_index_stride, num_rows), row_index_stride);
    groups[col_id * num_rowgroups + chunk_id] = *group;
  }
}

/**
 * @brief Get the buffer size and offsets of encoded statistics
 *
 * @param[in,out] groups Statistics merge groups
 * @param[in] statistics_count Number of statistics buffers
 *
 **/
constexpr unsigned int buffersize_reduction_dim = 32;
constexpr unsigned int buffersize_threads_per_block =
  buffersize_reduction_dim * buffersize_reduction_dim;
constexpr unsigned int pb_fld_hdrlen     = 1;
constexpr unsigned int pb_fld_hdrlen16   = 2;  // > 127-byte length
constexpr unsigned int pb_fld_hdrlen32   = 5;  // > 16KB length
constexpr unsigned int pb_fldlen_int64   = 10;
constexpr unsigned int pb_fldlen_float64 = 8;
constexpr unsigned int pb_fldlen_decimal = 40;  // Assume decimal2string fits in 40 characters
constexpr unsigned int pb_fldlen_bucket1 = 1 + pb_fldlen_int64;
constexpr unsigned int pb_fldlen_common  = 2 * pb_fld_hdrlen + pb_fldlen_int64;

__global__ void __launch_bounds__(buffersize_threads_per_block, 1)
  gpu_init_statistics_buffersize(statistics_merge_group *groups,
                                 const statistics_chunk *chunks,
                                 uint32_t statistics_count)
{
  __shared__ volatile uint32_t scratch_red[buffersize_reduction_dim];
  __shared__ volatile uint32_t stats_size;
  uint32_t tx = threadIdx.x;
  uint32_t ty = threadIdx.y;
  uint32_t t  = ty * buffersize_reduction_dim + tx;
  if (!t) { stats_size = 0; }
  __syncthreads();
  for (uint32_t start = 0; start < statistics_count; start += buffersize_threads_per_block) {
    uint32_t stats_len = 0, stats_pos;
    uint32_t idx       = start + t;
    if (idx < statistics_count) {
      const stats_column_desc *col = groups[idx].col;
      statistics_dtype dtype       = col->stats_dtype;
      switch (dtype) {
        case dtype_bool: stats_len = pb_fldlen_common + pb_fld_hdrlen + pb_fldlen_bucket1; break;
        case dtype_int8:
        case dtype_int16:
        case dtype_int32:
        case dtype_date32:
        case dtype_int64:
        case dtype_timestamp64:
          stats_len = pb_fldlen_common + pb_fld_hdrlen + 3 * (pb_fld_hdrlen + pb_fldlen_int64);
          break;
        case dtype_float32:
        case dtype_float64:
          stats_len = pb_fldlen_common + pb_fld_hdrlen + 3 * (pb_fld_hdrlen + pb_fldlen_float64);
          break;
        case dtype_decimal64:
        case dtype_decimal128:
          stats_len = pb_fldlen_common + pb_fld_hdrlen16 + 3 * (pb_fld_hdrlen + pb_fldlen_decimal);
          break;
        case dtype_string:
          stats_len = pb_fldlen_common + pb_fld_hdrlen32 + 3 * (pb_fld_hdrlen + pb_fldlen_int64) +
                      chunks[idx].min_value.str_val.length + chunks[idx].max_value.str_val.length;
          break;
        default: break;
      }
    }
    stats_pos = WarpReducePos32(stats_len, tx);
    if (tx == buffersize_reduction_dim - 1) { scratch_red[ty] = stats_pos; }
    __syncthreads();
    if (ty == 0) { scratch_red[tx] = WarpReducePos32(scratch_red[tx], tx); }
    __syncthreads();
    if (ty != 0) { stats_pos += scratch_red[ty - 1]; }
    stats_pos += stats_size;
    if (idx < statistics_count) {
      groups[idx].start_chunk = stats_pos - stats_len;
      groups[idx].num_chunks  = stats_len;
    }
    __syncthreads();
    if (t == buffersize_threads_per_block - 1) { stats_size = stats_pos; }
  }
}

struct stats_state_s {
  uint8_t *base;  ///< Output buffer start
  uint8_t *end;   ///< Output buffer end
  statistics_chunk chunk;
  statistics_merge_group group;
  stats_column_desc col;
  // ORC stats
  uint64_t numberOfValues;
  uint8_t hasNull;
};

/*
 * Protobuf encoding - see
 * https://developers.google.com/protocol-buffers/docs/encoding
 *
 */
// Protobuf varint encoding for unsigned int
__device__ inline uint8_t *pb_encode_uint(uint8_t *p, uint64_t v)
{
  while (v > 0x7f) {
    *p++ = ((uint32_t)v | 0x80);
    v >>= 7;
  }
  *p++ = v;
  return p;
}

// Protobuf field encoding for unsigned int
__device__ inline uint8_t *pb_put_uint(uint8_t *p, uint32_t id, uint64_t v)
{
  p[0] = id * 8 + PB_TYPE_VARINT;  // NOTE: Assumes id < 16
  return pb_encode_uint(p + 1, v);
}

// Protobuf field encoding for signed int
__device__ inline uint8_t *pb_put_int(uint8_t *p, uint32_t id, int64_t v)
{
  int64_t s = (v < 0);
  return pb_put_uint(p, id, (v ^ -s) * 2 + s);
}

// Protobuf field encoding for 'packed' unsigned int (single value)
__device__ inline uint8_t *pb_put_packed_uint(uint8_t *p, uint32_t id, uint64_t v)
{
  uint8_t *p2 = pb_encode_uint(p + 2, v);
  p[0]        = id * 8 + PB_TYPE_FIXEDLEN;
  p[1]        = static_cast<uint8_t>(p2 - (p + 2));
  return p2;
}

// Protobuf field encoding for binary/string
__device__ inline uint8_t *pb_put_binary(uint8_t *p, uint32_t id, const void *bytes, uint32_t len)
{
  p[0] = id * 8 + PB_TYPE_FIXEDLEN;
  p    = pb_encode_uint(p + 1, len);
  memcpy(p, bytes, len);
  return p + len;
}

// Protobuf field encoding for 64-bit raw encoding (double)
__device__ inline uint8_t *pb_put_fixed64(uint8_t *p, uint32_t id, const void *raw64)
{
  p[0] = id * 8 + PB_TYPE_FIXED64;
  memcpy(p + 1, raw64, 8);
  return p + 9;
}

/**
 * @brief Encode statistics in ORC protobuf format
 *
 * @param[in,out] groups Statistics merge groups
 * @param[in,out] chunks Statistics data
 * @param[in] statistics_count Number of statistics buffers
 *
 * ORC statistics format from https://orc.apache.org/specification/ORCv1/
 *
 * message ColumnStatistics {
 *  // the number of values
 *  optional uint64 numberOfValues = 1;
 *  // At most one of these has a value for any column
 *  optional IntegerStatistics intStatistics = 2;
 *  optional DoubleStatistics doubleStatistics = 3;
 *  optional StringStatistics stringStatistics = 4;
 *  optional BucketStatistics bucketStatistics = 5;
 *  optional DecimalStatistics decimalStatistics = 6;
 *  optional DateStatistics dateStatistics = 7;
 *  optional BinaryStatistics binaryStatistics = 8;
 *  optional TimestampStatistics timestampStatistics = 9;
 *  optional bool hasNull = 10;
 * }
 *
 **/
constexpr unsigned int encode_threads_per_chunk = 32;
constexpr unsigned int encode_chunks_per_block  = 4;
constexpr unsigned int encode_threads_per_block =
  encode_threads_per_chunk * encode_chunks_per_block;

__global__ void __launch_bounds__(encode_threads_per_block)
  gpu_encode_statistics(uint8_t *blob_bfr,
                        statistics_merge_group *groups,
                        const statistics_chunk *chunks,
                        uint32_t statistics_count)
{
  __shared__ __align__(8) stats_state_s state_g[encode_chunks_per_block];
  uint32_t t             = threadIdx.x;
  uint32_t idx           = blockIdx.x * encode_chunks_per_block + threadIdx.y;
  stats_state_s *const s = &state_g[threadIdx.y];

  // Encode and update actual bfr size
  if (idx < statistics_count && t == 0) {
    s->chunk           = chunks[idx];
    s->group           = groups[idx];
    s->col             = *(s->group.col);
    s->base            = blob_bfr + s->group.start_chunk;
    s->end             = blob_bfr + s->group.start_chunk + s->group.num_chunks;
    uint8_t *cur       = pb_put_uint(s->base, 1, s->chunk.non_nulls);
    uint8_t *fld_start = cur;
    switch (s->col.stats_dtype) {
      case dtype_int8:
      case dtype_int16:
      case dtype_int32:
      case dtype_int64:
        // intStatistics = 2
        // message IntegerStatistics {
        //  optional sint64 minimum = 1;
        //  optional sint64 maximum = 2;
        //  optional sint64 sum = 3;
        // }
        if (s->chunk.has_minmax || s->chunk.has_sum) {
          *cur = 2 * 8 + PB_TYPE_FIXEDLEN;
          cur += 2;
          if (s->chunk.has_minmax) {
            cur = pb_put_int(cur, 1, s->chunk.min_value.i_val);
            cur = pb_put_int(cur, 2, s->chunk.max_value.i_val);
          }
          if (s->chunk.has_sum) { cur = pb_put_int(cur, 3, s->chunk.sum.i_val); }
          fld_start[1] = cur - (fld_start + 2);
        }
        break;
      case dtype_float32:
      case dtype_float64:
        // doubleStatistics = 3
        // message DoubleStatistics {
        //  optional double minimum = 1;
        //  optional double maximum = 2;
        //  optional double sum = 3;
        // }
        if (s->chunk.has_minmax) {
          *cur = 3 * 8 + PB_TYPE_FIXEDLEN;
          cur += 2;
          cur          = pb_put_fixed64(cur, 1, &s->chunk.min_value.fp_val);
          cur          = pb_put_fixed64(cur, 2, &s->chunk.max_value.fp_val);
          fld_start[1] = cur - (fld_start + 2);
        }
        break;
      case dtype_string:
        // stringStatistics = 4
        // message StringStatistics {
        //  optional string minimum = 1;
        //  optional string maximum = 2;
        //  optional sint64 sum = 3; // sum will store the total length of all strings
        // }
        if (s->chunk.has_minmax && s->chunk.has_sum) {
          uint32_t sz = (pb_put_uint(cur, 3, s->chunk.sum.i_val) - cur) +
                        (pb_put_uint(cur, 1, s->chunk.min_value.str_val.length) - cur) +
                        (pb_put_uint(cur, 2, s->chunk.max_value.str_val.length) - cur) +
                        s->chunk.min_value.str_val.length + s->chunk.max_value.str_val.length;
          cur[0] = 4 * 8 + PB_TYPE_FIXEDLEN;
          cur    = pb_encode_uint(cur + 1, sz);
          cur    = pb_put_binary(
            cur, 1, s->chunk.min_value.str_val.ptr, s->chunk.min_value.str_val.length);
          cur = pb_put_binary(
            cur, 2, s->chunk.max_value.str_val.ptr, s->chunk.max_value.str_val.length);
          cur = pb_put_uint(cur, 3, s->chunk.sum.i_val);
        }
        break;
      case dtype_bool:
        // bucketStatistics = 5
        // message BucketStatistics {
        //  repeated uint64 count = 1 [packed=true];
        // }
        if (s->chunk.has_sum) {  // Sum is equal to the number of 'true' values
          cur[0]       = 5 * 8 + PB_TYPE_FIXEDLEN;
          cur          = pb_put_packed_uint(cur + 2, 1, s->chunk.sum.i_val);
          fld_start[1] = cur - (fld_start + 2);
        }
        break;
      case dtype_decimal64:
      case dtype_decimal128:
        // decimalStatistics = 6
        // message DecimalStatistics {
        //  optional string minimum = 1;
        //  optional string maximum = 2;
        //  optional string sum = 3;
        // }
        if (s->chunk.has_minmax) {
          // TODO: Decimal support (decimal min/max stored as strings)
        }
        break;
      case dtype_date32:
        // dateStatistics = 7
        // message DateStatistics { // min,max values saved as days since epoch
        //  optional sint32 minimum = 1;
        //  optional sint32 maximum = 2;
        // }
        if (s->chunk.has_minmax) {
          cur[0] = 7 * 8 + PB_TYPE_FIXEDLEN;
          cur += 2;
          cur          = pb_put_int(cur, 1, s->chunk.min_value.i_val);
          cur          = pb_put_int(cur, 2, s->chunk.max_value.i_val);
          fld_start[1] = cur - (fld_start + 2);
        }
        break;
      case dtype_timestamp64:
        // timestampStatistics = 9
        // message TimestampStatistics {
        //  optional sint64 minimum = 1; // min,max values saved as milliseconds since epoch
        //  optional sint64 maximum = 2;
        //  optional sint64 minimumUtc = 3; // min,max values saved as milliseconds since UNIX epoch
        //  optional sint64 maximumUtc = 4;
        // }
        if (s->chunk.has_minmax) {
          cur[0] = 9 * 8 + PB_TYPE_FIXEDLEN;
          cur += 2;
          cur          = pb_put_int(cur, 3, s->chunk.min_value.i_val);  // minimumUtc
          cur          = pb_put_int(cur, 4, s->chunk.max_value.i_val);  // maximumUtc
          fld_start[1] = cur - (fld_start + 2);
        }
        break;
      default: break;
    }
    groups[idx].num_chunks = static_cast<uint32_t>(cur - s->base);
  }
}

/**
 * @brief Launches kernels to initialize statistics collection
 *
 * @param[out] groups Statistics groups (rowgroup-level)
 * @param[in] cols Column descriptors
 * @param[in] num_columns Number of columns
 * @param[in] num_rowgroups Number of rowgroups
 * @param[in] row_index_stride Rowgroup size in rows
 * @param[in] stream CUDA stream to use, default 0
 */
void orc_init_statistics_groups(statistics_group *groups,
                                const stats_column_desc *cols,
                                uint32_t num_columns,
                                uint32_t num_rowgroups,
                                uint32_t row_index_stride,
                                hipStream_t stream)
{
  dim3 dim_grid((num_rowgroups + init_groups_per_block - 1) / init_groups_per_block, num_columns);
  dim3 dim_block(init_threads_per_group, init_groups_per_block);
  gpu_init_statistics_groups<<<dim_grid, dim_block, 0, stream>>>(
    groups, cols, num_columns, num_rowgroups, row_index_stride);
}

/**
 * @brief Launches kernels to return statistics buffer offsets and sizes
 *
 * @param[in,out] groups Statistics merge groups
 * @param[in] chunks Statistics chunks
 * @param[in] statistics_count Number of statistics buffers to encode
 * @param[in] stream CUDA stream to use, default 0
 */
void orc_init_statistics_buffersize(statistics_merge_group *groups,
                                    const statistics_chunk *chunks,
                                    uint32_t statistics_count,
                                    hipStream_t stream)
{
  dim3 dim_block(buffersize_reduction_dim, buffersize_reduction_dim);
  gpu_init_statistics_buffersize<<<1, dim_block, 0, stream>>>(groups, chunks, statistics_count);
}

/**
 * @brief Launches kernel to encode statistics in ORC protobuf format
 *
 * @param[out] blob_bfr Output buffer for statistics blobs
 * @param[in,out] groups Statistics merge groups
 * @param[in,out] chunks Statistics data
 * @param[in] statistics_count Number of statistics buffers
 */
void orc_encode_statistics(uint8_t *blob_bfr,
                           statistics_merge_group *groups,
                           const statistics_chunk *chunks,
                           uint32_t statistics_count,
                           hipStream_t stream)
{
  unsigned int num_blocks =
    (statistics_count + encode_chunks_per_block - 1) / encode_chunks_per_block;
  dim3 dim_block(encode_threads_per_chunk, encode_chunks_per_block);
  gpu_encode_statistics<<<num_blocks, dim_block, 0, stream>>>(
    blob_bfr, groups, chunks, statistics_count);
}

}  // namespace gpu
}  // namespace orc
}  // namespace io
}  // namespace cudf
