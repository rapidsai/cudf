#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2024, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "io/comp/gpuinflate.hpp"
#include "io/orc/reader_impl.hpp"
#include "io/orc/reader_impl_chunking.hpp"
#include "io/orc/reader_impl_helpers.hpp"
#include "io/utilities/hostdevice_span.hpp"

#include <cudf/detail/timezone.hpp>
#include <cudf/detail/utilities/integer_utils.hpp>
#include <cudf/utilities/error.hpp>

#include <rmm/device_buffer.hpp>
#include <rmm/exec_policy.hpp>

#include <thrust/binary_search.h>
#include <thrust/iterator/transform_iterator.h>
#include <thrust/scan.h>

#include <algorithm>
#include <tuple>

namespace cudf::io::orc::detail {

std::size_t gather_stream_info_and_column_desc(
  std::size_t stripe_id,
  std::size_t level,
  orc::StripeInformation const* stripeinfo,
  orc::StripeFooter const* stripefooter,
  host_span<int const> orc2gdf,
  host_span<orc::SchemaType const> types,
  bool use_index,
  bool apply_struct_map,
  int64_t* num_dictionary_entries,
  std::size_t* local_stream_order,
  std::vector<orc_stream_info>* stream_info,
  cudf::detail::hostdevice_2dvector<gpu::ColumnDesc>* chunks)
{
  CUDF_EXPECTS((stream_info == nullptr) ^ (chunks == nullptr),
               "Either stream_info or chunks must be provided, but not both.");

  std::size_t src_offset = 0;
  std::size_t dst_offset = 0;

  auto const get_stream_index_type = [](orc::StreamKind kind) {
    switch (kind) {
      case orc::DATA: return gpu::CI_DATA;
      case orc::LENGTH:
      case orc::SECONDARY: return gpu::CI_DATA2;
      case orc::DICTIONARY_DATA: return gpu::CI_DICTIONARY;
      case orc::PRESENT: return gpu::CI_PRESENT;
      case orc::ROW_INDEX: return gpu::CI_INDEX;
      default:
        // Skip this stream as it's not strictly required
        return gpu::CI_NUM_STREAMS;
    }
  };

  for (auto const& stream : stripefooter->streams) {
    if (!stream.column_id || *stream.column_id >= orc2gdf.size()) {
      // Ignore reading this stream from source.
      CUDF_LOG_WARN("Unexpected stream in the input ORC source. The stream will be ignored.");
      src_offset += stream.length;
      continue;
    }

    auto const column_id = *stream.column_id;
    auto col             = orc2gdf[column_id];

    if (col == -1 and apply_struct_map) {
      // A struct-type column has no data itself, but rather child columns
      // for each of its fields. There is only a PRESENT stream, which
      // needs to be included for the reader.
      auto const schema_type = types[column_id];
      if (!schema_type.subtypes.empty() && schema_type.kind == orc::STRUCT &&
          stream.kind == orc::PRESENT) {
        for (auto const& idx : schema_type.subtypes) {
          auto const child_idx = (idx < orc2gdf.size()) ? orc2gdf[idx] : -1;
          if (child_idx >= 0) {
            col = child_idx;
            if (chunks) {
              auto& chunk                     = (*chunks)[stripe_id][col];
              chunk.strm_id[gpu::CI_PRESENT]  = *local_stream_order;
              chunk.strm_len[gpu::CI_PRESENT] = stream.length;
            }
          }
        }
      }
    } else if (col != -1) {
      if (chunks) {
        if (src_offset >= stripeinfo->indexLength || use_index) {
          auto const index_type = get_stream_index_type(stream.kind);
          if (index_type < gpu::CI_NUM_STREAMS) {
            auto& chunk                = (*chunks)[stripe_id][col];
            chunk.strm_id[index_type]  = *local_stream_order;
            chunk.strm_len[index_type] = stream.length;
            // NOTE: skip_count field is temporarily used to track the presence of index streams
            chunk.skip_count |= 1 << index_type;

            if (index_type == gpu::CI_DICTIONARY) {
              chunk.dictionary_start = *num_dictionary_entries;
              chunk.dict_len         = stripefooter->columns[column_id].dictionarySize;
              *num_dictionary_entries +=
                static_cast<int64_t>(stripefooter->columns[column_id].dictionarySize);
            }
          }
        }

        (*local_stream_order)++;
      } else {  // chunks == nullptr
        stream_info->emplace_back(
          orc_stream_info{stripeinfo->offset + src_offset,
                          dst_offset,
                          stream.length,
                          stream_source_info{stripe_id, level, column_id, stream.kind}});
      }

      dst_offset += stream.length;
    }
    src_offset += stream.length;
  }

  return dst_offset;
}

template <typename T>
std::vector<range> find_splits(host_span<T const> cumulative_sizes,
                               std::size_t total_count,
                               std::size_t size_limit)
{
  CUDF_EXPECTS(size_limit > 0, "Invalid size limit", std::invalid_argument);

  std::vector<range> splits;
  std::size_t cur_count{0};
  int64_t cur_pos{0};
  std::size_t cur_cumulative_size{0};

  [[maybe_unused]] std::size_t cur_cumulative_rows{0};

  auto const start = thrust::make_transform_iterator(
    cumulative_sizes.begin(),
    [&](auto const& size) { return size.size_bytes - cur_cumulative_size; });
  auto const end = start + cumulative_sizes.size();

  while (cur_count < total_count) {
    int64_t split_pos = static_cast<int64_t>(
      thrust::distance(start, thrust::lower_bound(thrust::seq, start + cur_pos, end, size_limit)));

    // If we're past the end, or if the returned range has size exceeds the given size limit,
    // move back one position.
    if (split_pos >= static_cast<int64_t>(cumulative_sizes.size()) ||
        (cumulative_sizes[split_pos].size_bytes > cur_cumulative_size + size_limit)) {
      split_pos--;
    }

    if constexpr (std::is_same_v<T, cumulative_size_and_row>) {
      // Similarly, while the returned range has total number of rows exceeds column size limit,
      // move back one position.
      while (split_pos > 0 && cumulative_sizes[split_pos].num_rows >
                                cur_cumulative_rows +
                                  static_cast<std::size_t>(std::numeric_limits<size_type>::max())) {
        split_pos--;
      }
    }

    // In case we have moved back too much in the steps above, far beyond the last split point, that
    // means we could not find any range that has size fits within the given size limit.
    // In such situations, we need to move forward until we move pass the last output range.
    while (split_pos < (static_cast<int64_t>(cumulative_sizes.size()) - 1) &&
           (split_pos < 0 || cumulative_sizes[split_pos].count <= cur_count)) {
      split_pos++;
    }

    auto const start_count = cur_count;
    cur_count              = cumulative_sizes[split_pos].count;
    splits.emplace_back(range{start_count, cur_count});
    cur_pos             = split_pos;
    cur_cumulative_size = cumulative_sizes[split_pos].size_bytes;

    if constexpr (std::is_same_v<T, cumulative_size_and_row>) {
      cur_cumulative_rows = cumulative_sizes[split_pos].num_rows;
    }
  }

  // If the last range has size smaller than `merge_threshold` the size of the second last one,
  // merge it with the second last one.
  // This is to prevent having the last range too small.
  if (splits.size() > 1) {
    double constexpr merge_threshold = 0.15;
    if (auto const last = splits.back(), second_last = splits[splits.size() - 2];
        last.size() <= static_cast<std::size_t>(merge_threshold * second_last.size())) {
      splits.pop_back();
      splits.back().end = last.end;
    }
  }

  return splits;
}

// Since `find_splits` is a template function, we need to explicitly instantiate it so it can be
// used outside of this TU.
template std::vector<range> find_splits<cumulative_size>(host_span<cumulative_size const> sizes,
                                                         std::size_t total_count,
                                                         std::size_t size_limit);
template std::vector<range> find_splits<cumulative_size_and_row>(
  host_span<cumulative_size_and_row const> sizes, std::size_t total_count, std::size_t size_limit);

// In this step, the metadata of all stripes in the data sources is parsed, and information about
// data streams of the selected columns in all stripes are generated. If the reader has a data
// read limit, sizes of these streams are used to split the list of all stripes into multiple
// subsets, each of which will be loaded into memory in the `load_next_stripe_data()` step. These
// subsets are computed such that memory usage will be kept to be around a fixed size limit.
void reader_impl::preprocess_file(read_mode mode)
{
  if (_file_itm_data.global_preprocessed) { return; }
  _file_itm_data.global_preprocessed = true;

  //
  // Load stripes' metadata:
  //
  std::tie(
    _file_itm_data.rows_to_skip, _file_itm_data.rows_to_read, _file_itm_data.selected_stripes) =
    _metadata.select_stripes(
      _options.selected_stripes, _options.skip_rows, _options.num_read_rows, _stream);
  if (!_file_itm_data.has_data()) { return; }

  CUDF_EXPECTS(
    mode == read_mode::CHUNKED_READ ||
      _file_itm_data.rows_to_read <= static_cast<int64_t>(std::numeric_limits<size_type>::max()),
    "READ_ALL mode does not support reading number of rows more than cudf's column size limit. "
    "For reading large number of rows, please use chunked_reader.",
    std::overflow_error);

  auto const& selected_stripes = _file_itm_data.selected_stripes;
  auto const num_total_stripes = selected_stripes.size();
  auto const num_levels        = _selected_columns.num_levels();

  // Set up table for converting timestamp columns from local to UTC time
  _file_itm_data.tz_table = [&] {
    auto const has_timestamp_column = std::any_of(
      _selected_columns.levels.cbegin(), _selected_columns.levels.cend(), [&](auto const& col_lvl) {
        return std::any_of(col_lvl.cbegin(), col_lvl.cend(), [&](auto const& col_meta) {
          return _metadata.get_col_type(col_meta.id).kind == TypeKind::TIMESTAMP;
        });
      });

    return has_timestamp_column ? cudf::detail::make_timezone_transition_table(
                                    {}, selected_stripes[0].stripe_footer->writerTimezone, _stream)
                                : std::make_unique<cudf::table>();
  }();

  //
  // Pre allocate necessary memory for data processed in the other reading steps:
  //
  auto& stripe_data_read_ranges = _file_itm_data.stripe_data_read_ranges;
  stripe_data_read_ranges.resize(num_total_stripes);

  auto& lvl_stripe_data          = _file_itm_data.lvl_stripe_data;
  auto& lvl_stripe_sizes         = _file_itm_data.lvl_stripe_sizes;
  auto& lvl_stream_info          = _file_itm_data.lvl_stream_info;
  auto& lvl_stripe_stream_ranges = _file_itm_data.lvl_stripe_stream_ranges;
  auto& lvl_column_types         = _file_itm_data.lvl_column_types;
  auto& lvl_nested_cols          = _file_itm_data.lvl_nested_cols;

  lvl_stripe_data.resize(num_levels);
  lvl_stripe_sizes.resize(num_levels);
  lvl_stream_info.resize(num_levels);
  lvl_stripe_stream_ranges.resize(num_levels);
  lvl_column_types.resize(num_levels);
  lvl_nested_cols.resize(num_levels);
  _out_buffers.resize(num_levels);

  auto& read_info = _file_itm_data.data_read_info;
  auto& col_meta  = *_col_meta;

  //
  // Collect columns' types:
  //
  for (std::size_t level = 0; level < num_levels; ++level) {
    lvl_stripe_sizes[level].resize(num_total_stripes);
    lvl_stripe_stream_ranges[level].resize(num_total_stripes);

    // Association between each ORC column and its cudf::column
    col_meta.orc_col_map.emplace_back(_metadata.get_num_cols(), -1);

    auto const& columns_level = _selected_columns.levels[level];
    size_type col_id{0};

    for (auto const& col : columns_level) {
      // Map each ORC column to its column
      col_meta.orc_col_map[level][col.id] = col_id++;

      auto const col_type =
        to_cudf_type(_metadata.get_col_type(col.id).kind,
                     _options.use_np_dtypes,
                     _options.timestamp_type.id(),
                     to_cudf_decimal_type(_options.decimal128_columns, _metadata, col.id));
      CUDF_EXPECTS(col_type != type_id::EMPTY, "Unknown type");

      auto& column_types = lvl_column_types[level];
      auto& nested_cols  = lvl_nested_cols[level];

      if (col_type == type_id::DECIMAL32 or col_type == type_id::DECIMAL64 or
          col_type == type_id::DECIMAL128) {
        // sign of the scale is changed since cuDF follows c++ libraries like CNL
        // which uses negative scaling, but liborc and other libraries
        // follow positive scaling.
        auto const scale =
          -static_cast<size_type>(_metadata.get_col_type(col.id).scale.value_or(0));
        column_types.emplace_back(col_type, scale);
      } else {
        column_types.emplace_back(col_type);
      }

      // Map each ORC column to its column.
      if (col_type == type_id::LIST or col_type == type_id::STRUCT) {
        nested_cols.emplace_back(col);
      }
    }

    // Try to reserve some memory, but the final size is unknown,
    // since each column may have more than one stream.
    auto const num_columns = columns_level.size();
    lvl_stream_info[level].reserve(num_total_stripes * num_columns);
    if (read_info.capacity() < num_total_stripes * num_columns) {
      read_info.reserve(num_total_stripes * num_columns);
    }
  }

  //
  // Collect all data streams' information:
  //

  // Load all stripes if we are in READ_ALL mode or there is no read limit.
  auto const load_all_stripes =
    mode == read_mode::READ_ALL || _chunk_read_data.pass_read_limit == 0;

  // Accumulate data size for data streams in each stripe, used for chunking.
  // This will be used only for CHUNKED_READ mode when there is a read limit.
  // Otherwise, we do not need this since we just load all stripes.
  cudf::detail::hostdevice_vector<cumulative_size> total_stripe_sizes(
    load_all_stripes ? std::size_t{0} : num_total_stripes, _stream);

  for (std::size_t stripe_global_idx = 0; stripe_global_idx < num_total_stripes;
       ++stripe_global_idx) {
    auto const& stripe       = selected_stripes[stripe_global_idx];
    auto const stripe_info   = stripe.stripe_info;
    auto const stripe_footer = stripe.stripe_footer;

    std::size_t this_stripe_size{0};
    auto const last_read_size = read_info.size();
    for (std::size_t level = 0; level < num_levels; ++level) {
      auto& stream_info = _file_itm_data.lvl_stream_info[level];

      auto stream_level_count = stream_info.size();
      auto const stripe_level_size =
        gather_stream_info_and_column_desc(stripe_global_idx,
                                           level,
                                           stripe_info,
                                           stripe_footer,
                                           col_meta.orc_col_map[level],
                                           _metadata.get_types(),
                                           false,  // use_index,
                                           level == 0,
                                           nullptr,  // num_dictionary_entries
                                           nullptr,  // local_stream_order
                                           &stream_info,
                                           nullptr  // chunks
        );

      auto const is_stripe_data_empty = stripe_level_size == 0;
      CUDF_EXPECTS(not is_stripe_data_empty or stripe_info->indexLength == 0,
                   "Invalid index rowgroup stream data");

      lvl_stripe_sizes[level][stripe_global_idx] = stripe_level_size;
      this_stripe_size += stripe_level_size;

      // Range of the streams in `stream_info` corresponding to this stripe at the current level.
      lvl_stripe_stream_ranges[level][stripe_global_idx] =
        range{stream_level_count, stream_info.size()};

      // Coalesce consecutive streams into one read.
      while (not is_stripe_data_empty and stream_level_count < stream_info.size()) {
        auto const d_dst  = stream_info[stream_level_count].dst_pos;
        auto const offset = stream_info[stream_level_count].offset;
        auto len          = stream_info[stream_level_count].length;
        stream_level_count++;

        while (stream_level_count < stream_info.size() &&
               stream_info[stream_level_count].offset == offset + len) {
          len += stream_info[stream_level_count].length;
          stream_level_count++;
        }
        read_info.emplace_back(stream_data_read_info{offset,
                                                     d_dst,
                                                     len,
                                                     static_cast<std::size_t>(stripe.source_idx),
                                                     stripe_global_idx,
                                                     level});
      }
    }  // end loop level

    if (!load_all_stripes) { total_stripe_sizes[stripe_global_idx] = {1, this_stripe_size}; }

    // Range of all stream reads in `read_info` corresponding to this stripe, in all levels.
    stripe_data_read_ranges[stripe_global_idx] = range{last_read_size, read_info.size()};
  }

  //
  // Split range of all stripes into subranges that can be loaded separately while maintaining
  // the memory usage under the given pass limit:
  //

  // Load range is reset to start from the first position in `load_stripe_ranges`.
  _chunk_read_data.curr_load_stripe_range = 0;

  if (load_all_stripes) {
    _chunk_read_data.load_stripe_ranges = {range{0UL, num_total_stripes}};
    return;
  }

  // Compute the prefix sum of stripes' data sizes.
  total_stripe_sizes.host_to_device_async(_stream);
  thrust::inclusive_scan(rmm::exec_policy_nosync(_stream),
                         total_stripe_sizes.d_begin(),
                         total_stripe_sizes.d_end(),
                         total_stripe_sizes.d_begin(),
                         cumulative_size_plus{});
  total_stripe_sizes.device_to_host_sync(_stream);

  auto const load_limit = [&] {
    auto const tmp = static_cast<std::size_t>(_chunk_read_data.pass_read_limit *
                                              chunk_read_data::load_limit_ratio);
    // Make sure not to pass 0 byte limit (due to round-off) to `find_splits`.
    return std::max(tmp, 1UL);
  }();

  _chunk_read_data.load_stripe_ranges =
    find_splits<cumulative_size>(total_stripe_sizes, num_total_stripes, load_limit);
}

// If there is a data read limit, only a subset of stripes are read at a time such that
// their total data size does not exceed a fixed size limit. Then, the data is probed to
// estimate its uncompressed sizes, which are in turn used to split that stripe subset into
// smaller subsets, each of which to be decompressed and decoded in the next step
// `decompress_and_decode_stripes()`. This is to ensure that loading data from data sources
// together with decompression and decoding will be capped around the given data read limit.
void reader_impl::load_next_stripe_data(read_mode mode)
{
  if (!_file_itm_data.has_data()) { return; }

  auto const load_stripe_range =
    _chunk_read_data.load_stripe_ranges[_chunk_read_data.curr_load_stripe_range++];
  auto const stripe_start = load_stripe_range.begin;
  auto const stripe_count = load_stripe_range.size();

  auto& lvl_stripe_data = _file_itm_data.lvl_stripe_data;
  auto const num_levels = _selected_columns.num_levels();

  // Prepare the buffer to read raw data onto.
  for (std::size_t level = 0; level < num_levels; ++level) {
    auto& stripe_data = lvl_stripe_data[level];
    stripe_data.resize(stripe_count);

    for (std::size_t idx = 0; idx < stripe_count; ++idx) {
      auto const stripe_size = _file_itm_data.lvl_stripe_sizes[level][idx + stripe_start];
      stripe_data[idx]       = rmm::device_buffer(
        cudf::util::round_up_safe(stripe_size, BUFFER_PADDING_MULTIPLE), _stream);
    }
  }

  //
  // Load stripe data into memory:
  //

  // If we load data from sources into host buffers, we need to transfer (async) data to device
  // memory. Such host buffers need to be kept alive until we sync the transfers.
  std::vector<std::unique_ptr<cudf::io::datasource::buffer>> host_read_buffers;

  // If we load data directly from sources into device memory, the loads are also async.
  // Thus, we need to make sure to sync all them at the end.
  std::vector<std::pair<std::future<std::size_t>, std::size_t>> device_read_tasks;

  // Range of the read info (offset, length) to read for the current being loaded stripes.
  auto const [read_begin, read_end] =
    merge_selected_ranges(_file_itm_data.stripe_data_read_ranges, load_stripe_range);

  for (auto read_idx = read_begin; read_idx < read_end; ++read_idx) {
    auto const& read_info = _file_itm_data.data_read_info[read_idx];
    auto const source_ptr = _metadata.per_file_metadata[read_info.source_idx].source;
    auto const dst_base   = static_cast<uint8_t*>(
      lvl_stripe_data[read_info.level][read_info.stripe_idx - stripe_start].data());

    if (source_ptr->is_device_read_preferred(read_info.length)) {
      device_read_tasks.push_back(
        std::pair(source_ptr->device_read_async(
                    read_info.offset, read_info.length, dst_base + read_info.dst_pos, _stream),
                  read_info.length));

    } else {
      auto buffer = source_ptr->host_read(read_info.offset, read_info.length);
      CUDF_EXPECTS(buffer->size() == read_info.length, "Unexpected discrepancy in bytes read.");
      CUDF_CUDA_TRY(hipMemcpyAsync(dst_base + read_info.dst_pos,
                                    buffer->data(),
                                    read_info.length,
                                    hipMemcpyDefault,
                                    _stream.value()));
      host_read_buffers.emplace_back(std::move(buffer));
    }
  }

  if (host_read_buffers.size() > 0) {  // if there was host read
    _stream.synchronize();
    host_read_buffers.clear();  // its data was copied to device memory after stream sync
  }
  for (auto& task : device_read_tasks) {  // if there was device read
    CUDF_EXPECTS(task.first.get() == task.second, "Unexpected discrepancy in bytes read.");
  }

  // Compute number of rows in the loading stripes.
  auto const num_loading_rows = std::accumulate(
    _file_itm_data.selected_stripes.begin() + stripe_start,
    _file_itm_data.selected_stripes.begin() + stripe_start + stripe_count,
    std::size_t{0},
    [](std::size_t count, const auto& stripe) { return count + stripe.stripe_info->numberOfRows; });

  // Decoding range needs to be reset to start from the first position in `decode_stripe_ranges`.
  _chunk_read_data.curr_decode_stripe_range = 0;

  // The cudf's column size limit.
  auto constexpr column_size_limit =
    static_cast<std::size_t>(std::numeric_limits<size_type>::max());

  // Decode all loaded stripes if there is no read limit, or if we are in READ_ALL mode,
  // and the number of loading rows is less than the column size limit.
  // In theory, we should just decode 'enough' stripes for output one table chunk, instead of
  // decoding all stripes like this, for better load-balancing and reduce memory usage.
  // However, we do not have any good way to know how many stripes are 'enough'.
  if ((mode == read_mode::READ_ALL || _chunk_read_data.pass_read_limit == 0) &&
      // In addition to read limit, we also need to check if the total number of
      // rows in the loaded stripes exceeds the column size limit.
      // If that is the case, we cannot decode all stripes at once into a cudf table.
      num_loading_rows <= column_size_limit) {
    _chunk_read_data.decode_stripe_ranges = {load_stripe_range};
    return;
  }

  // From here, we have reading mode that is either:
  // - CHUNKED_READ without read limit but the number of reading rows exceeds column size limit, or
  // - CHUNKED_READ with a pass read limit.
  // READ_ALL mode with number of rows more than cudf's column size limit should be handled early in
  // `preprocess_file`. We just check again to make sure such situations never happen here.
  CUDF_EXPECTS(
    mode != read_mode::READ_ALL,
    "READ_ALL mode does not support reading number of rows more than cudf's column size limit.");

  // This is the post-processing step after we've done with splitting `load_stripe_range` into
  // `decode_stripe_ranges`.
  auto const add_range_offset = [stripe_start](std::vector<range>& new_ranges) {
    // The split ranges always start from zero.
    // We need to change these ranges to start from `stripe_start` which are the correct subranges
    // of the current loaded stripe range.
    for (auto& range : new_ranges) {
      range.begin += stripe_start;
      range.end += stripe_start;
    }
  };

  // Optimized code path when we do not have any read limit but the number of rows in the
  // loaded stripes exceeds column size limit.
  // Note that the values `max_uncompressed_size` for each stripe are not computed here.
  // Instead, they will be computed on the fly during decoding to avoid the overhead of
  // storing and retrieving from memory.
  if (_chunk_read_data.pass_read_limit == 0 && num_loading_rows > column_size_limit) {
    std::vector<cumulative_size_and_row> cumulative_stripe_rows(stripe_count);
    std::size_t rows{0};

    for (std::size_t idx = 0; idx < stripe_count; ++idx) {
      auto const& stripe     = _file_itm_data.selected_stripes[idx + stripe_start];
      auto const stripe_info = stripe.stripe_info;
      rows += stripe_info->numberOfRows;

      // We will split stripe ranges based only on stripes' number of rows, not data size.
      // Thus, we override the cumulative `size_bytes` using the prefix sum of rows in stripes and
      // will use the column size limit as the split size limit.
      cumulative_stripe_rows[idx] =
        cumulative_size_and_row{idx + 1UL /*count*/, rows /*size_bytes*/, rows};
    }

    _chunk_read_data.decode_stripe_ranges =
      find_splits<cumulative_size_and_row>(cumulative_stripe_rows, stripe_count, column_size_limit);
    add_range_offset(_chunk_read_data.decode_stripe_ranges);
    return;
  }

  //
  // Split range of loaded stripes into subranges that can be decoded separately such that the
  // memory usage is maintained around the given limit:
  //

  // This is for estimating the decompressed sizes of the loaded stripes.
  cudf::detail::hostdevice_vector<cumulative_size_and_row> stripe_decomp_sizes(stripe_count,
                                                                               _stream);

  // Fill up the `cumulative_size_and_row` array with initial values.
  // Note: `hostdevice_vector::begin()` mirrors `std::vector::data()` using incorrect API name.
  for (std::size_t idx = 0; idx < stripe_count; ++idx) {
    auto const& stripe     = _file_itm_data.selected_stripes[idx + stripe_start];
    auto const stripe_info = stripe.stripe_info;
    stripe_decomp_sizes[idx] =
      cumulative_size_and_row{1UL /*count*/, 0UL /*size_bytes*/, stripe_info->numberOfRows};
  }

  auto& compinfo_map = _file_itm_data.compinfo_map;
  compinfo_map.clear();  // clear cache of the last load

  // For parsing decompression data.
  // We create an array that is large enough to use for all levels, thus only need to allocate
  // memory once.
  auto hd_compinfo = [&] {
    std::size_t max_num_streams{0};
    if (_metadata.per_file_metadata[0].ps.compression != orc::NONE) {
      // Find the maximum number of streams in all levels of the loaded stripes.
      for (std::size_t level = 0; level < num_levels; ++level) {
        auto const stream_range =
          merge_selected_ranges(_file_itm_data.lvl_stripe_stream_ranges[level], load_stripe_range);
        max_num_streams = std::max(max_num_streams, stream_range.size());
      }
    }
    return cudf::detail::hostdevice_vector<gpu::CompressedStreamInfo>(max_num_streams, _stream);
  }();

  for (std::size_t level = 0; level < num_levels; ++level) {
    auto const& stream_info = _file_itm_data.lvl_stream_info[level];
    auto const num_columns  = _selected_columns.levels[level].size();

    auto& stripe_data = lvl_stripe_data[level];
    if (stripe_data.empty()) { continue; }

    // Range of all streams in the loaded stripes.
    auto const stream_range =
      merge_selected_ranges(_file_itm_data.lvl_stripe_stream_ranges[level], load_stripe_range);

    if (_metadata.per_file_metadata[0].ps.compression != orc::NONE) {
      auto const& decompressor = *_metadata.per_file_metadata[0].decompressor;

      auto compinfo = cudf::detail::hostdevice_span<gpu::CompressedStreamInfo>(
        hd_compinfo.begin(), hd_compinfo.d_begin(), stream_range.size());
      for (auto stream_idx = stream_range.begin; stream_idx < stream_range.end; ++stream_idx) {
        auto const& info = stream_info[stream_idx];
        auto const dst_base =
          static_cast<uint8_t const*>(stripe_data[info.source.stripe_idx - stripe_start].data());
        compinfo[stream_idx - stream_range.begin] =
          gpu::CompressedStreamInfo(dst_base + info.dst_pos, info.length);
      }

      // Estimate the uncompressed data.
      compinfo.host_to_device_async(_stream);
      gpu::ParseCompressedStripeData(compinfo.device_ptr(),
                                     compinfo.size(),
                                     decompressor.GetBlockSize(),
                                     decompressor.GetLog2MaxCompressionRatio(),
                                     _stream);
      compinfo.device_to_host_sync(_stream);

      for (auto stream_idx = stream_range.begin; stream_idx < stream_range.end; ++stream_idx) {
        auto const& info           = stream_info[stream_idx];
        auto const stream_compinfo = compinfo[stream_idx - stream_range.begin];

        // Cache these parsed numbers so they can be reused in the decompression/decoding step.
        compinfo_map[info.source] = {stream_compinfo.num_compressed_blocks,
                                     stream_compinfo.num_uncompressed_blocks,
                                     stream_compinfo.max_uncompressed_size};
        stripe_decomp_sizes[info.source.stripe_idx - stripe_start].size_bytes +=
          stream_compinfo.max_uncompressed_size;
      }

    } else {  // no decompression
      // Set decompression sizes equal to the input sizes.
      for (auto stream_idx = stream_range.begin; stream_idx < stream_range.end; ++stream_idx) {
        auto const& info = stream_info[stream_idx];
        stripe_decomp_sizes[info.source.stripe_idx - stripe_start].size_bytes += info.length;
      }
    }
  }  // end loop level

  // Compute the prefix sum of stripe data sizes and rows.
  stripe_decomp_sizes.host_to_device_async(_stream);
  thrust::inclusive_scan(rmm::exec_policy_nosync(_stream),
                         stripe_decomp_sizes.d_begin(),
                         stripe_decomp_sizes.d_end(),
                         stripe_decomp_sizes.d_begin(),
                         cumulative_size_plus{});
  stripe_decomp_sizes.device_to_host_sync(_stream);

  auto const decode_limit = [&] {
    auto const tmp = static_cast<std::size_t>(_chunk_read_data.pass_read_limit *
                                              chunk_read_data::decompress_and_decode_limit_ratio);
    // Make sure not to pass 0 byte limit to `find_splits`.
    return std::max(tmp, 1UL);
  }();

  _chunk_read_data.decode_stripe_ranges =
    find_splits<cumulative_size_and_row>(stripe_decomp_sizes, stripe_count, decode_limit);

  add_range_offset(_chunk_read_data.decode_stripe_ranges);
}

}  // namespace cudf::io::orc::detail
