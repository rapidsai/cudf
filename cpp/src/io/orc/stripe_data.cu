#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <io/utilities/block_utils.cuh>
#include "orc_common.h"
#include "orc_gpu.h"

#define LOG2_BYTESTREAM_BFRSZ 13  // Must be able to handle 512x 8-byte values

#define BYTESTREAM_BFRSZ (1 << LOG2_BYTESTREAM_BFRSZ)
#define BYTESTREAM_BFRMASK32 ((BYTESTREAM_BFRSZ - 1) >> 2)
// TODO: Should be more efficient with 512 threads per block and circular queue for values
#define LOG2_NWARPS 5  // Log2 of number of warps per threadblock
#define LOG2_NTHREADS (LOG2_NWARPS + 5)
#define NWARPS (1 << LOG2_NWARPS)
#define NTHREADS (1 << LOG2_NTHREADS)
#define ROWDEC_BFRSZ \
  (NTHREADS + 128)  // Add some margin to look ahead to future rows in case there are many zeroes

#define IS_RLEv1(encoding_mode) ((encoding_mode) < DIRECT_V2)
#define IS_RLEv2(encoding_mode) ((encoding_mode) >= DIRECT_V2)
#define IS_DICTIONARY(encoding_mode) ((encoding_mode)&1)

namespace cudf {
namespace io {
namespace orc {
namespace gpu {
static __device__ __constant__ int64_t kORCTimeToUTC =
  1420070400;  // Seconds from January 1st, 1970 to January 1st, 2015

struct int128_s {
  uint64_t lo;
  int64_t hi;
};

struct orc_bytestream_s {
  const uint8_t *base;
  uint32_t pos;
  uint32_t len;
  uint32_t fill_pos;
  uint32_t fill_count;
  union {
    uint8_t u8[BYTESTREAM_BFRSZ];
    uint32_t u32[BYTESTREAM_BFRSZ >> 2];
    uint2 u64[BYTESTREAM_BFRSZ >> 3];
  } buf;
};

struct orc_rlev1_state_s {
  uint32_t num_runs;
  uint32_t num_vals;
  int32_t run_data[NWARPS * 12];  // (delta << 24) | (count << 16) | (first_val)
};

struct orc_rlev2_state_s {
  uint32_t num_runs;
  uint32_t num_vals;
  union {
    uint32_t u32[NWARPS];
    uint64_t u64[NWARPS];
  } baseval;
  uint16_t m2_pw_byte3[NWARPS];
  int64_t delta[NWARPS];
  uint16_t runs_loc[NTHREADS];
};

struct orc_byterle_state_s {
  uint32_t num_runs;
  uint32_t num_vals;
  uint32_t runs_loc[NWARPS];
  uint32_t runs_pos[NWARPS];
};

struct orc_rowdec_state_s {
  uint32_t nz_count;
  uint32_t last_row[NWARPS];
  uint32_t row[ROWDEC_BFRSZ];  // 0=skip, >0: row position relative to cur_row
};

struct orc_strdict_state_s {
  uint2 *local_dict;
  uint32_t dict_pos;
  uint32_t dict_len;
};

struct orc_nulldec_state_s {
  uint32_t row;
  uint32_t null_count[NWARPS];
};

struct orc_datadec_state_s {
  uint32_t cur_row;             // starting row of current batch
  uint32_t end_row;             // ending row of this chunk (start_row + num_rows)
  uint32_t max_vals;            // max # of non-zero values to decode in this batch
  uint32_t nrows;               // # of rows in current batch (up to NTHREADS)
  uint32_t buffered_count;      // number of buffered values in the secondary data stream
  uint32_t tz_num_entries;      // number of entries in timezone table
  uint32_t tz_dst_cycle;        // number of entries in timezone daylight savings cycle
  int64_t first_tz_transition;  // first transition in timezone table
  int64_t last_tz_transition;   // last transition in timezone table
  int64_t utc_epoch;            // kORCTimeToUTC - gmtOffset
  RowGroup index;
};

struct orcdec_state_s {
  ColumnDesc chunk;
  orc_bytestream_s bs;
  orc_bytestream_s bs2;
  int is_string;
  union {
    orc_strdict_state_s dict;
    orc_nulldec_state_s nulls;
    orc_datadec_state_s data;
  } top;
  union {
    orc_rlev1_state_s rlev1;
    orc_rlev2_state_s rlev2;
    orc_byterle_state_s rle8;
    orc_rowdec_state_s rowdec;
  } u;
  union {
    uint8_t u8[NTHREADS * 8];
    uint32_t u32[NTHREADS * 2];
    int32_t i32[NTHREADS * 2];
    uint64_t u64[NTHREADS];
    int64_t i64[NTHREADS];
  } vals;
};

/**
 * @brief Initializes byte stream, modifying length and start position to keep the read pointer
 *8-byte aligned Assumes that the address range [start_address & ~7, (start_address + len - 1) | 7]
 *is valid
 *
 * @param[in] bs Byte stream input
 * @param[in] base Pointer to raw byte stream data
 * @param[in] len Stream length in bytes
 *
 **/
static __device__ void bytestream_init(volatile orc_bytestream_s *bs,
                                       const uint8_t *base,
                                       uint32_t len)
{
  uint32_t pos   = static_cast<uint32_t>(7 & reinterpret_cast<size_t>(base));
  bs->base       = base - pos;
  bs->pos        = (len > 0) ? pos : 0;
  bs->len        = (len + pos + 7) & ~7;
  bs->fill_pos   = 0;
  bs->fill_count = min(bs->len, BYTESTREAM_BFRSZ) >> 3;
}

/**
 * @brief Increment the read position, returns number of 64-bit slots to fill
 *
 * @param[in] bs Byte stream input
 * @param[in] bytes_consumed Number of bytes that were consumed
 *
 **/
static __device__ void bytestream_flush_bytes(volatile orc_bytestream_s *bs,
                                              uint32_t bytes_consumed)
{
  uint32_t pos     = bs->pos;
  uint32_t len     = bs->len;
  uint32_t pos_new = min(pos + bytes_consumed, len);
  bs->pos          = pos_new;
  pos              = min(pos + BYTESTREAM_BFRSZ, len);
  pos_new          = min(pos_new + BYTESTREAM_BFRSZ, len);
  bs->fill_pos     = pos;
  bs->fill_count   = (pos_new >> 3) - (pos >> 3);
}

/**
 * @brief Refill the byte stream buffer
 *
 * @param[in] bs Byte stream input
 * @param[in] t thread id
 *
 **/
static __device__ void bytestream_fill(orc_bytestream_s *bs, int t)
{
  int count = bs->fill_count;
  if (t < count) {
    int pos8 = (bs->fill_pos >> 3) + t;
    bs->buf.u64[pos8 & ((BYTESTREAM_BFRSZ >> 3) - 1)] =
      (reinterpret_cast<const uint2 *>(bs->base))[pos8];
  }
}

/**
 * @brief Read a byte from the byte stream (byte aligned)
 *
 * @param[in] bs Byte stream input
 * @param[in] pos Position in byte stream
 * @return byte
 *
 **/
inline __device__ uint8_t bytestream_readbyte(volatile orc_bytestream_s *bs, int pos)
{
  return bs->buf.u8[pos & (BYTESTREAM_BFRSZ - 1)];
}

/**
 * @brief Read 32 bits from a byte stream (little endian, byte aligned)
 *
 * @param[in] bs Byte stream input
 * @param[in] pos Position in byte stream
 * @result bits
 *
 **/
inline __device__ uint32_t bytestream_readu32(volatile orc_bytestream_s *bs, int pos)
{
  uint32_t a = bs->buf.u32[(pos & (BYTESTREAM_BFRSZ - 1)) >> 2];
  uint32_t b = bs->buf.u32[((pos + 4) & (BYTESTREAM_BFRSZ - 1)) >> 2];
  return __funnelshift_r(a, b, (pos & 3) * 8);
}

/**
 * @brief Read 64 bits from a byte stream (little endian, byte aligned)
 *
 * @param[in] bs Byte stream input
 * @param[in] pos Position in byte stream
 * @param[in] numbits number of bits
 * @return bits
 *
 **/
inline __device__ uint64_t bytestream_readu64(volatile orc_bytestream_s *bs, int pos)
{
  uint32_t a    = bs->buf.u32[(pos & (BYTESTREAM_BFRSZ - 1)) >> 2];
  uint32_t b    = bs->buf.u32[((pos + 4) & (BYTESTREAM_BFRSZ - 1)) >> 2];
  uint32_t c    = bs->buf.u32[((pos + 8) & (BYTESTREAM_BFRSZ - 1)) >> 2];
  uint32_t lo32 = __funnelshift_r(a, b, (pos & 3) * 8);
  uint32_t hi32 = __funnelshift_r(b, c, (pos & 3) * 8);
  uint64_t v    = hi32;
  v <<= 32;
  v |= lo32;
  return v;
}

/**
 * @brief Read up to 32-bits from a byte stream (big endian)
 *
 * @param[in] bs Byte stream input
 * @param[in] bitpos Position in byte stream
 * @param[in] numbits number of bits
 * @return decoded value
 *
 **/
inline __device__ uint32_t bytestream_readbits(volatile orc_bytestream_s *bs,
                                               int bitpos,
                                               uint32_t numbits)
{
  int idx    = bitpos >> 5;
  uint32_t a = __byte_perm(bs->buf.u32[(idx + 0) & BYTESTREAM_BFRMASK32], 0, 0x0123);
  uint32_t b = __byte_perm(bs->buf.u32[(idx + 1) & BYTESTREAM_BFRMASK32], 0, 0x0123);
  return __funnelshift_l(b, a, bitpos & 0x1f) >> (32 - numbits);
}

/**
 * @brief Read up to 64-bits from a byte stream (big endian)
 *
 * @param[in] bs Byte stream input
 * @param[in] bitpos Position in byte stream
 * @param[in] numbits number of bits
 * @return decoded value
 *
 **/
inline __device__ uint64_t bytestream_readbits64(volatile orc_bytestream_s *bs,
                                                 int bitpos,
                                                 uint32_t numbits)
{
  int idx       = bitpos >> 5;
  uint32_t a    = __byte_perm(bs->buf.u32[(idx + 0) & BYTESTREAM_BFRMASK32], 0, 0x0123);
  uint32_t b    = __byte_perm(bs->buf.u32[(idx + 1) & BYTESTREAM_BFRMASK32], 0, 0x0123);
  uint32_t c    = __byte_perm(bs->buf.u32[(idx + 2) & BYTESTREAM_BFRMASK32], 0, 0x0123);
  uint32_t hi32 = __funnelshift_l(b, a, bitpos & 0x1f);
  uint32_t lo32 = __funnelshift_l(c, b, bitpos & 0x1f);
  uint64_t v    = hi32;
  v <<= 32;
  v |= lo32;
  v >>= (64 - numbits);
  return v;
}

/**
 * @brief Decode a big-endian unsigned 32-bit value
 *
 * @param[in] bs Byte stream input
 * @param[in] bitpos Position in byte stream
 * @param[in] numbits number of bits
 * @param[out] result decoded value
 *
 **/
inline __device__ void bytestream_readbe(volatile orc_bytestream_s *bs,
                                         int bitpos,
                                         uint32_t numbits,
                                         uint32_t &result)
{
  result = bytestream_readbits(bs, bitpos, numbits);
}

/**
 * @brief Decode a big-endian signed 32-bit value
 *
 * @param[in] bs Byte stream input
 * @param[in] bitpos Position in byte stream
 * @param[in] numbits number of bits
 * @param[out] result decoded value
 *
 **/
inline __device__ void bytestream_readbe(volatile orc_bytestream_s *bs,
                                         int bitpos,
                                         uint32_t numbits,
                                         int32_t &result)
{
  uint32_t u = bytestream_readbits(bs, bitpos, numbits);
  result     = (int32_t)((u >> 1u) ^ -(int32_t)(u & 1));
}

/**
 * @brief Decode a big-endian unsigned 64-bit value
 *
 * @param[in] bs Byte stream input
 * @param[in] bitpos Position in byte stream
 * @param[in] numbits number of bits
 * @param[out] result decoded value
 *
 **/
inline __device__ void bytestream_readbe(volatile orc_bytestream_s *bs,
                                         int bitpos,
                                         uint32_t numbits,
                                         uint64_t &result)
{
  result = bytestream_readbits64(bs, bitpos, numbits);
}

/**
 * @brief Decode a big-endian signed 64-bit value
 *
 * @param[in] bs Byte stream input
 * @param[in] bitpos Position in byte stream
 * @param[in] numbits number of bits
 * @param[out] result decoded value
 *
 **/
inline __device__ void bytestream_readbe(volatile orc_bytestream_s *bs,
                                         int bitpos,
                                         uint32_t numbits,
                                         int64_t &result)
{
  uint64_t u = bytestream_readbits64(bs, bitpos, numbits);
  result     = (int64_t)((u >> 1u) ^ -(int64_t)(u & 1));
}

/**
 * @brief Return the length of a base-128 varint
 *
 * @param[in] bs Byte stream input
 * @param[in] pos Position in circular byte stream buffer
 * @return length of varint in bytes
 **/
template <class T>
inline __device__ uint32_t varint_length(volatile orc_bytestream_s *bs, int pos)
{
  if (bytestream_readbyte(bs, pos) > 0x7f) {
    uint32_t next32 = bytestream_readu32(bs, pos + 1);
    uint32_t zbit   = __ffs((~next32) & 0x80808080);
    if (sizeof(T) <= 4 || zbit) {
      return 1 + (zbit >> 3);  // up to 5x7 bits
    } else {
      next32 = bytestream_readu32(bs, pos + 5);
      zbit   = __ffs((~next32) & 0x80808080);
      if (zbit) {
        return 5 + (zbit >> 3);  // up to 9x7 bits
      } else if ((sizeof(T) <= 8) || (bytestream_readbyte(bs, pos + 9) <= 0x7f)) {
        return 10;  // up to 70 bits
      } else {
        uint64_t next64 = bytestream_readu64(bs, pos + 10);
        zbit            = __ffsll((~next64) & 0x8080808080808080ull);
        if (zbit) {
          return 10 + (zbit >> 3);  // Up to 18x7 bits (126)
        } else {
          return 19;  // Up to 19x7 bits (133)
        }
      }
    }
  } else {
    return 1;
  }
}

/**
 * @brief Decodes a base-128 varint
 *
 * @param[in] bs Byte stream input
 * @param[in] pos Position in circular byte stream buffer
 * @param[in] result Unpacked value
 * @return new position in byte stream buffer
 **/
template <class T>
inline __device__ int decode_base128_varint(volatile orc_bytestream_s *bs, int pos, T &result)
{
  uint32_t v = bytestream_readbyte(bs, pos++);
  if (v > 0x7f) {
    uint32_t b = bytestream_readbyte(bs, pos++);
    v          = (v & 0x7f) | (b << 7);
    if (b > 0x7f) {
      b = bytestream_readbyte(bs, pos++);
      v = (v & 0x3fff) | (b << 14);
      if (b > 0x7f) {
        b = bytestream_readbyte(bs, pos++);
        v = (v & 0x1fffff) | (b << 21);
        if (b > 0x7f) {
          b = bytestream_readbyte(bs, pos++);
          v = (v & 0x0fffffff) | (b << 28);
          if (sizeof(T) > 4) {
            uint32_t lo = v;
            uint64_t hi;
            v = b >> 4;
            if (b > 0x7f) {
              b = bytestream_readbyte(bs, pos++);
              v = (v & 7) | (b << 3);
              if (b > 0x7f) {
                b = bytestream_readbyte(bs, pos++);
                v = (v & 0x3ff) | (b << 10);
                if (b > 0x7f) {
                  b = bytestream_readbyte(bs, pos++);
                  v = (v & 0x1ffff) | (b << 17);
                  if (b > 0x7f) {
                    b = bytestream_readbyte(bs, pos++);
                    v = (v & 0xffffff) | (b << 24);
                    if (b > 0x7f) {
                      pos++;  // last bit is redundant (extra byte implies bit63 is 1)
                    }
                  }
                }
              }
            }
            hi = v;
            hi <<= 32;
            result = hi | lo;
            return pos;
          }
        }
      }
    }
  }
  result = v;
  return pos;
}

/**
 * @brief Decodes a signed int128 encoded as base-128 varint (used for decimals)
 **/
inline __device__ int128_s decode_varint128(volatile orc_bytestream_s *bs, int pos)
{
  uint32_t b        = bytestream_readbyte(bs, pos++);
  int64_t sign_mask = -(int32_t)(b & 1);
  uint64_t v        = (b >> 1) & 0x3f;
  uint32_t bitpos   = 6;
  uint64_t lo       = v;
  uint64_t hi       = 0;
  while (b > 0x7f && bitpos < 128) {
    b = bytestream_readbyte(bs, pos++);
    v |= ((uint64_t)(b & 0x7f)) << (bitpos & 0x3f);
    if (bitpos == 62) {  // 6 + 7 * 8 = 62
      lo = v;
      v  = (b & 0x7f) >> 2;  // 64 - 62
    }
    bitpos += 7;
  }
  if (bitpos >= 64) {
    hi = v;
  } else {
    lo = v;
  }
  return {(uint64_t)(lo ^ sign_mask), (int64_t)(hi ^ sign_mask)};
}

/**
 * @brief Decodes an unsigned 32-bit varint
 **/
inline __device__ int decode_varint(volatile orc_bytestream_s *bs, int pos, uint32_t &result)
{
  uint32_t u;
  pos    = decode_base128_varint<uint32_t>(bs, pos, u);
  result = u;
  return pos;
}

/**
 * @brief Decodes an unsigned 64-bit varint
 **/
inline __device__ int decode_varint(volatile orc_bytestream_s *bs, int pos, uint64_t &result)
{
  uint64_t u;
  pos    = decode_base128_varint<uint64_t>(bs, pos, u);
  result = u;
  return pos;
}

/**
 * @brief Signed version of 32-bit decode_varint
 **/
inline __device__ int decode_varint(volatile orc_bytestream_s *bs, int pos, int32_t &result)
{
  uint32_t u;
  pos    = decode_base128_varint<uint32_t>(bs, pos, u);
  result = (int32_t)((u >> 1u) ^ -(int32_t)(u & 1));
  return pos;
}

/**
 * @brief Signed version of 64-bit decode_varint
 **/
inline __device__ int decode_varint(volatile orc_bytestream_s *bs, int pos, int64_t &result)
{
  uint64_t u;
  pos    = decode_base128_varint<uint64_t>(bs, pos, u);
  result = (int64_t)((u >> 1u) ^ -(int64_t)(u & 1));
  return pos;
}

/**
 * @brief In-place conversion from lengths to positions
 *
 * @param[in] vals input values
 * @param[in] numvals number of values
 * @param[in] t thread id
 *
 * @return number of values decoded
 **/
template <class T>
inline __device__ void lengths_to_positions(volatile T *vals, uint32_t numvals, unsigned int t)
{
  for (uint32_t n = 1; n < numvals; n <<= 1) {
    __syncthreads();
    if ((t & n) && (t < numvals)) vals[t] += vals[(t & ~n) | (n - 1)];
  }
}

/**
 * @brief ORC Integer RLEv1 decoding
 *
 * @param[in] bs input byte stream
 * @param[in] rle RLE state
 * @param[in] vals buffer for output values (uint32_t, int32_t, uint64_t or int64_t)
 * @param[in] maxvals maximum number of values to decode
 * @param[in] t thread id
 *
 * @return number of values decoded
 **/
template <class T>
static __device__ uint32_t Integer_RLEv1(
  orc_bytestream_s *bs, volatile orc_rlev1_state_s *rle, volatile T *vals, uint32_t maxvals, int t)
{
  uint32_t numvals, numruns;
  if (t == 0) {
    uint32_t maxpos  = min(bs->len, bs->pos + (BYTESTREAM_BFRSZ - 8u));
    uint32_t lastpos = bs->pos;
    numvals = numruns = 0;
    // Find the length and start location of each run
    while (numvals < maxvals && numruns < NWARPS * 12) {
      uint32_t pos = lastpos;
      uint32_t n   = bytestream_readbyte(bs, pos++);
      if (n <= 0x7f) {
        // Run
        int32_t delta;
        n = n + 3;
        if (numvals + n > maxvals) break;
        delta         = bytestream_readbyte(bs, pos++);
        vals[numvals] = pos & 0xffff;
        pos += varint_length<T>(bs, pos);
        if (pos > maxpos) break;
        rle->run_data[numruns++] = (delta << 24) | (n << 16) | numvals;
        numvals += n;
      } else {
        // Literals
        uint32_t i;
        n = 0x100 - n;
        if (numvals + n > maxvals) break;
        i = 0;
        do {
          vals[numvals + i] = pos & 0xffff;
          pos += varint_length<T>(bs, pos);
        } while (++i < n);
        if (pos > maxpos) break;
        numvals += n;
      }
      lastpos = pos;
    }
    rle->num_runs = numruns;
    rle->num_vals = numvals;
    bytestream_flush_bytes(bs, lastpos - bs->pos);
  }
  __syncthreads();
  // Expand the runs
  numruns = rle->num_runs;
  if (numruns > 0) {
    int r  = t >> 5;
    int tr = t & 0x1f;
    for (uint32_t run = r; run < numruns; run += NWARPS) {
      int32_t run_data = rle->run_data[run];
      int n            = (run_data >> 16) & 0xff;
      int delta        = run_data >> 24;
      uint32_t base    = run_data & 0x3ff;
      uint32_t pos     = vals[base] & 0xffff;
      for (int i = 1 + tr; i < n; i += 32) { vals[base + i] = ((delta * i) << 16) | pos; }
    }
    __syncthreads();
  }
  numvals = rle->num_vals;
  // Decode individual 32-bit varints
  if (t < numvals) {
    int32_t pos   = vals[t];
    int32_t delta = pos >> 16;
    T v;
    decode_varint(bs, pos, v);
    vals[t] = v + delta;
  }
  __syncthreads();
  return numvals;
}

/**
 * @brief Maps the RLEv2 5-bit length code to 6-bit length
 *
 **/
static const __device__ __constant__ uint8_t kRLEv2_W[32] = {
  1,  2,  3,  4,  5,  6,  7,  8,  9,  10, 11, 12, 13, 14, 15, 16,
  17, 18, 19, 20, 21, 22, 23, 24, 26, 28, 30, 32, 40, 48, 56, 64};

/**
 * @brief Maps the RLEv2 patch size (pw + pgw) to number of bits
 *
 * Patch size (in bits) is only allowed to be from the below set. If `pw + pgw == 34` then the size
 * of the patch in the file is the smallest size in the set that can fit 34 bits i.e.
 * `ClosestFixedBitsMap[34] == 40`
 *
 * @see https://github.com/apache/orc/commit/9faf7f5147a7bc69
 */
static const __device__ __constant__ uint8_t ClosestFixedBitsMap[65] = {
  1,  1,  2,  3,  4,  5,  6,  7,  8,  9,  10, 11, 12, 13, 14, 15, 16, 17, 18, 19, 20, 21,
  22, 23, 24, 26, 26, 28, 28, 30, 30, 32, 32, 40, 40, 40, 40, 40, 40, 40, 40, 48, 48, 48,
  48, 48, 48, 48, 48, 56, 56, 56, 56, 56, 56, 56, 56, 64, 64, 64, 64, 64, 64, 64, 64};

/**
 * @brief ORC Integer RLEv2 decoding
 *
 * @param[in] bs input byte stream
 * @param[in] rle RLE state
 * @param[in] vals buffer for output values (uint32_t, int32_t, uint64_t or int64_t)
 * @param[in] maxvals maximum number of values to decode
 * @param[in] t thread id
 *
 * @return number of values decoded
 **/
template <class T>
static __device__ uint32_t Integer_RLEv2(
  orc_bytestream_s *bs, volatile orc_rlev2_state_s *rle, volatile T *vals, uint32_t maxvals, int t)
{
  uint32_t numvals, numruns;
  int r, tr;

  if (t == 0) {
    uint32_t maxpos  = min(bs->len, bs->pos + (BYTESTREAM_BFRSZ - 8u));
    uint32_t lastpos = bs->pos;
    numvals = numruns = 0;
    // Find the length and start location of each run
    while (numvals < maxvals) {
      uint32_t pos   = lastpos;
      uint32_t byte0 = bytestream_readbyte(bs, pos++);
      uint32_t n, l;
      int mode               = byte0 >> 6;
      rle->runs_loc[numruns] = numvals;
      vals[numvals]          = lastpos;
      if (mode == 0) {
        // 00lllnnn: short repeat encoding
        l = 1 + ((byte0 >> 3) & 7);  // 1 to 8 bytes
        n = 3 + (byte0 & 7);         // 3 to 10 values
      } else {
        l = kRLEv2_W[(byte0 >> 1) & 0x1f];
        n = 1 + ((byte0 & 1) << 8) + bytestream_readbyte(bs, pos++);
        if (mode == 1) {
          // 01wwwwwn.nnnnnnnn: direct encoding
          l = (l * n + 7) >> 3;
        } else if (mode == 2) {
          // 10wwwwwn.nnnnnnnn.xxxxxxxx.yyyyyyyy: patched base encoding
          uint32_t byte2      = bytestream_readbyte(bs, pos++);
          uint32_t byte3      = bytestream_readbyte(bs, pos++);
          uint32_t bw         = 1 + (byte2 >> 5);        // base value width, 1 to 8 bytes
          uint32_t pw         = kRLEv2_W[byte2 & 0x1f];  // patch width, 1 to 64 bits
          uint32_t pgw        = 1 + (byte3 >> 5);        // patch gap width, 1 to 8 bits
          uint32_t pgw_pw_len = ClosestFixedBitsMap[min(pw + pgw, 64u)];  // ceiled patch width
          uint32_t pll        = byte3 & 0x1f;                             // patch list length
          l                   = (l * n + 7) >> 3;
          l += bw;
          l += (pll * (pgw_pw_len) + 7) >> 3;
        } else {
          // 11wwwwwn.nnnnnnnn.<base>.<delta>: delta encoding
          uint32_t deltapos = varint_length<T>(bs, pos);
          deltapos += varint_length<T>(bs, pos + deltapos);
          l = (l > 1 && n > 2) ? (l * (n - 2) + 7) >> 3 : 0;
          l += deltapos;
        }
      }
      if (numvals + n > maxvals) break;
      pos += l;
      if (pos > maxpos) break;
      lastpos = pos;
      numvals += n;
      numruns++;
    }
    rle->num_vals = numvals;
    rle->num_runs = numruns;
    bytestream_flush_bytes(bs, lastpos - bs->pos);
  }
  __syncthreads();
  // Process the runs, 1 warp per run
  numruns = rle->num_runs;
  r       = t >> 5;
  tr      = t & 0x1f;
  for (uint32_t run = r; run < numruns; run += NWARPS) {
    uint32_t base, pos, w, n;
    int mode;
    if (tr == 0) {
      uint32_t byte0;
      base  = rle->runs_loc[run];
      pos   = vals[base];
      byte0 = bytestream_readbyte(bs, pos++);
      mode  = byte0 >> 6;
      if (mode == 0) {
        T baseval;
        // 00lllnnn: short repeat encoding
        w = 8 + (byte0 & 0x38);  // 8 to 64 bits
        n = 3 + (byte0 & 7);     // 3 to 10 values
        bytestream_readbe(bs, pos * 8, w, baseval);
        if (sizeof(T) <= 4) {
          rle->baseval.u32[r] = baseval;
        } else {
          rle->baseval.u64[r] = baseval;
        }
      } else {
        w = kRLEv2_W[(byte0 >> 1) & 0x1f];
        n = 1 + ((byte0 & 1) << 8) + bytestream_readbyte(bs, pos++);
        if (mode > 1) {
          if (mode == 2) {
            // Patched base
            uint32_t byte2 = bytestream_readbyte(bs, pos++);
            uint32_t byte3 = bytestream_readbyte(bs, pos++);
            uint32_t bw    = 1 + (byte2 >> 5);        // base value width, 1 to 8 bytes
            uint32_t pw    = kRLEv2_W[byte2 & 0x1f];  // patch width, 1 to 64 bits
            if (sizeof(T) <= 4) {
              uint32_t baseval, mask;
              bytestream_readbe(bs, pos * 8, bw * 8, baseval);
              mask                = (1 << (bw * 8 - 1)) - 1;
              rle->baseval.u32[r] = (baseval > mask) ? (-(int32_t)(baseval & mask)) : baseval;
            } else {
              uint64_t baseval, mask;
              bytestream_readbe(bs, pos * 8, bw * 8, baseval);
              mask = 1;
              mask <<= (bw * 8) - 1;
              mask -= 1;
              rle->baseval.u64[r] = (baseval > mask) ? (-(int64_t)(baseval & mask)) : baseval;
            }
            rle->m2_pw_byte3[r] = (pw << 8) | byte3;
            pos += bw;
          } else {
            T baseval;
            int64_t delta;
            // Delta
            pos = decode_varint(bs, pos, baseval);
            if (sizeof(T) <= 4) {
              rle->baseval.u32[r] = baseval;
            } else {
              rle->baseval.u64[r] = baseval;
            }
            pos           = decode_varint(bs, pos, delta);
            rle->delta[r] = delta;
          }
        }
      }
    }
    base = SHFL0(base);
    mode = SHFL0(mode);
    pos  = SHFL0(pos);
    n    = SHFL0(n);
    w    = SHFL0(w);
    for (uint32_t i = tr; i < n; i += 32) {
      if (sizeof(T) <= 4) {
        if (mode == 0) {
          vals[base + i] = rle->baseval.u32[r];
        } else if (mode == 1) {
          T v;
          bytestream_readbe(bs, pos * 8 + i * w, w, v);
          vals[base + i] = v;
        } else if (mode == 2) {
          uint32_t ofs   = bytestream_readbits(bs, pos * 8 + i * w, w);
          vals[base + i] = rle->baseval.u32[r] + ofs;
        } else {
          int64_t delta = rle->delta[r];
          if (w > 1 && i > 1) {
            int32_t delta_s = (delta < 0) ? -1 : 0;
            vals[base + i] =
              (bytestream_readbits(bs, pos * 8 + (i - 2) * w, w) ^ delta_s) - delta_s;
          } else {
            vals[base + i] = (i == 0) ? 0 : static_cast<uint32_t>(delta);
          }
        }
      } else {
        if (mode == 0) {
          vals[base + i] = rle->baseval.u64[r];
        } else if (mode == 1) {
          T v;
          bytestream_readbe(bs, pos * 8 + i * w, w, v);
          vals[base + i] = v;
        } else if (mode == 2) {
          uint64_t ofs   = bytestream_readbits64(bs, pos * 8 + i * w, w);
          vals[base + i] = rle->baseval.u64[r] + ofs;
        } else {
          int64_t delta = rle->delta[r], ofs;
          if (w > 1 && i > 1) {
            int64_t delta_s = (delta < 0) ? -1 : 0;
            ofs = (bytestream_readbits64(bs, pos * 8 + (i - 2) * w, w) ^ delta_s) - delta_s;
          } else {
            ofs = (i == 0) ? 0 : delta;
          }
          vals[base + i] = ofs;
        }
      }
    }
    SYNCWARP();
    // Patch values
    if (mode == 2) {
      uint32_t pw_byte3 = rle->m2_pw_byte3[r];
      uint32_t pw       = pw_byte3 >> 8;
      uint32_t pgw      = 1 + ((pw_byte3 >> 5) & 7);  // patch gap width, 1 to 8 bits
      uint32_t pll      = pw_byte3 & 0x1f;            // patch list length
      if (pll != 0) {
        uint32_t pgw_pw_len = ClosestFixedBitsMap[min(pw + pgw, 64u)];
        uint64_t patch_pos64 =
          (tr < pll) ? bytestream_readbits64(
                         bs, pos * 8 + ((n * w + 7) & ~7) + tr * (pgw_pw_len), pgw_pw_len)
                     : 0;
        uint32_t patch_pos;
        T patch = 1;
        patch <<= pw;
        patch = (patch - 1) & (T)patch_pos64;
        patch <<= w;
        patch_pos = (uint32_t)(patch_pos64 >> pw);
        for (uint32_t k = 1; k < pll; k <<= 1) {
          uint32_t tmp = SHFL(patch_pos, (tr & ~k) | (k - 1));
          patch_pos += (tr & k) ? tmp : 0;
        }
        if (tr < pll && patch_pos < n) { vals[base + patch_pos] += patch; }
      }
    }
    SYNCWARP();
    if (mode == 3) {
      T baseval;
      for (uint32_t i = 1; i < n; i <<= 1) {
        SYNCWARP();
        for (uint32_t j = tr; j < n; j += 32) {
          if (j & i) vals[base + j] += vals[base + ((j & ~i) | (i - 1))];
        }
      }
      if (sizeof(T) <= 4)
        baseval = rle->baseval.u32[r];
      else
        baseval = rle->baseval.u64[r];
      for (uint32_t j = tr; j < n; j += 32) { vals[base + j] += baseval; }
    }
  }
  __syncthreads();
  return rle->num_vals;
}

/**
 * @brief Reads 32 booleans as a packed 32-bit value
 *
 * @param[in] vals 32-bit array of values (little-endian)
 * @param[in] bitpos bit position
 *
 * @return 32-bit value
 **/
inline __device__ uint32_t rle8_read_bool32(volatile uint32_t *vals, uint32_t bitpos)
{
  uint32_t a = vals[(bitpos >> 5) + 0];
  uint32_t b = vals[(bitpos >> 5) + 1];
  a          = __byte_perm(a, 0, 0x0123);
  b          = __byte_perm(b, 0, 0x0123);
  return __brev(__funnelshift_l(b, a, bitpos));
}

/**
 * @brief ORC Byte RLE decoding
 *
 * @param[in] bs Input byte stream
 * @param[in] rle RLE state
 * @param[in] vals output buffer for decoded 8-bit values
 * @param[in] maxvals Maximum number of values to decode
 * @param[in] t thread id
 *
 * @return number of values decoded
 **/
static __device__ uint32_t Byte_RLE(orc_bytestream_s *bs,
                                    volatile orc_byterle_state_s *rle,
                                    volatile uint8_t *vals,
                                    uint32_t maxvals,
                                    int t)
{
  uint32_t numvals, numruns;
  int r, tr;
  if (t == 0) {
    uint32_t maxpos  = min(bs->len, bs->pos + (BYTESTREAM_BFRSZ - 8u));
    uint32_t lastpos = bs->pos;
    numvals = numruns = 0;
    // Find the length and start location of each run
    while (numvals < maxvals && numruns < NWARPS) {
      uint32_t pos           = lastpos, n;
      rle->runs_pos[numruns] = pos;
      rle->runs_loc[numruns] = numvals;
      n                      = bytestream_readbyte(bs, pos++);
      if (n <= 0x7f) {
        // Run
        n = n + 3;
        pos++;
      } else {
        // Literals
        n = 0x100 - n;
        pos += n;
      }
      if (pos > maxpos || numvals + n > maxvals) { break; }
      numruns++;
      numvals += n;
      lastpos = pos;
    }
    rle->num_runs = numruns;
    rle->num_vals = numvals;
    bytestream_flush_bytes(bs, lastpos - bs->pos);
  }
  __syncthreads();
  numruns = rle->num_runs;
  r       = t >> 5;
  tr      = t & 0x1f;
  for (int run = r; run < numruns; run += NWARPS) {
    uint32_t pos = rle->runs_pos[run];
    uint32_t loc = rle->runs_loc[run];
    uint32_t n   = bytestream_readbyte(bs, pos++);
    uint32_t literal_mask;
    if (n <= 0x7f) {
      literal_mask = 0;
      n += 3;
    } else {
      literal_mask = ~0;
      n            = 0x100 - n;
    }
    for (uint32_t i = tr; i < n; i += 32) {
      vals[loc + i] = bytestream_readbyte(bs, pos + (i & literal_mask));
    }
  }
  __syncthreads();
  return rle->num_vals;
}

/**
 * @brief Powers of 10
 *
 **/
static const __device__ __constant__ double kPow10[40] = {
  1.0,   1.e1,  1.e2,  1.e3,  1.e4,  1.e5,  1.e6,  1.e7,  1.e8,  1.e9,  1.e10, 1.e11, 1.e12, 1.e13,
  1.e14, 1.e15, 1.e16, 1.e17, 1.e18, 1.e19, 1.e20, 1.e21, 1.e22, 1.e23, 1.e24, 1.e25, 1.e26, 1.e27,
  1.e28, 1.e29, 1.e30, 1.e31, 1.e32, 1.e33, 1.e34, 1.e35, 1.e36, 1.e37, 1.e38, 1.e39,
};

static const __device__ __constant__ int64_t kPow5i[28] = {1,
                                                           5,
                                                           25,
                                                           125,
                                                           625,
                                                           3125,
                                                           15625,
                                                           78125,
                                                           390625,
                                                           1953125,
                                                           9765625,
                                                           48828125,
                                                           244140625,
                                                           1220703125,
                                                           6103515625ll,
                                                           30517578125ll,
                                                           152587890625ll,
                                                           762939453125ll,
                                                           3814697265625ll,
                                                           19073486328125ll,
                                                           95367431640625ll,
                                                           476837158203125ll,
                                                           2384185791015625ll,
                                                           11920928955078125ll,
                                                           59604644775390625ll,
                                                           298023223876953125ll,
                                                           1490116119384765625ll,
                                                           7450580596923828125ll};

/**
 * @brief ORC Decimal decoding (unbounded base-128 varints)
 *
 * @param[in] bs Input byte stream
 * @param[in,out] vals on input: scale from secondary stream, on output: value
 * @param[in] numvals Number of values to decode
 * @param[in] t thread id
 *
 * @return number of values decoded
 *
 **/
static __device__ int Decode_Decimals(orc_bytestream_s *bs,
                                      volatile orc_byterle_state_s *scratch,
                                      volatile int64_t *vals,
                                      int val_scale,
                                      int numvals,
                                      int col_scale,
                                      int t)
{
  uint32_t num_vals_read = 0;
  // Iterates till `numvals` are read or there is nothing to read once the
  // stream has reached its end, and can't read anything more.
  while (num_vals_read != numvals) {
    if (t == 0) {
      uint32_t maxpos  = min(bs->len, bs->pos + (BYTESTREAM_BFRSZ - 8u));
      uint32_t lastpos = bs->pos;
      uint32_t n;
      for (n = num_vals_read; n < numvals; n++) {
        uint32_t pos = lastpos;
        pos += varint_length<uint4>(bs, pos);
        if (pos > maxpos) break;
        *reinterpret_cast<volatile int32_t *>(&vals[n]) = lastpos;
        lastpos                                         = pos;
      }
      scratch->num_vals = n;
      bytestream_flush_bytes(bs, lastpos - bs->pos);
    }
    __syncthreads();
    uint32_t num_vals_to_read = scratch->num_vals;
    if (t >= num_vals_read and t < num_vals_to_read) {
      int pos    = *reinterpret_cast<volatile int32_t *>(&vals[t]);
      int128_s v = decode_varint128(bs, pos);

      if (col_scale & ORC_DECIMAL2FLOAT64_SCALE) {
        double f      = Int128ToDouble_rn(v.lo, v.hi);
        int32_t scale = (t < numvals) ? val_scale : 0;
        if (scale >= 0)
          reinterpret_cast<volatile double *>(vals)[t] = f / kPow10[min(scale, 39)];
        else
          reinterpret_cast<volatile double *>(vals)[t] = f * kPow10[min(-scale, 39)];
      } else {
        int32_t scale = (t < numvals) ? (col_scale & ~ORC_DECIMAL2FLOAT64_SCALE) - val_scale : 0;
        if (scale >= 0) {
          scale   = min(scale, 27);
          vals[t] = ((int64_t)v.lo * kPow5i[scale]) << scale;
        } else  // if (scale < 0)
        {
          bool is_negative = (v.hi < 0);
          uint64_t hi = v.hi, lo = v.lo;
          scale = min(-scale, 27);
          if (is_negative) {
            hi = (~hi) + (lo == 0);
            lo = (~lo) + 1;
          }
          lo = (lo >> (uint32_t)scale) | ((uint64_t)hi << (64 - scale));
          hi >>= (int32_t)scale;
          if (hi != 0) {
            // Use intermediate float
            lo = __double2ull_rn(Int128ToDouble_rn(lo, hi) / __ll2double_rn(kPow5i[scale]));
            hi = 0;
          } else {
            lo /= kPow5i[scale];
          }
          vals[t] = (is_negative) ? -(int64_t)lo : (int64_t)lo;
        }
      }
    }
    // There is nothing to read, so break
    if (num_vals_read == num_vals_to_read) break;

    // Update number of values read (This contains values of previous iteration)
    num_vals_read = num_vals_to_read;

    // Have to wait till all threads have copied data
    __syncthreads();
    if (num_vals_read != numvals) {
      bytestream_fill(bs, t);
      __syncthreads();
      if (t == 0) {
        // Needs to be reset since bytestream has been filled
        bs->fill_count = 0;
      }
    }
    // Adding to get all threads in sync before next read
    __syncthreads();
  }
  return num_vals_read;
}

/**
 * @brief Decoding NULLs and builds string dictionary index tables
 *
 * @param[in] chunks ColumnDesc device array [stripe][column]
 * @param[in] global_dictionary Global dictionary device array
 * @param[in] num_columns Number of columns
 * @param[in] num_stripes Number of stripes
 * @param[in] max_num_rows Maximum number of rows to load
 * @param[in] first_row Crop all rows below first_row
 *
 **/
// blockDim {NTHREADS,1,1}
extern "C" __global__ void __launch_bounds__(NTHREADS)
  gpuDecodeNullsAndStringDictionaries(ColumnDesc *chunks,
                                      DictionaryEntry *global_dictionary,
                                      uint32_t num_columns,
                                      uint32_t num_stripes,
                                      size_t max_num_rows,
                                      size_t first_row)
{
  __shared__ __align__(16) orcdec_state_s state_g;

  orcdec_state_s *const s = &state_g;
  bool is_nulldec         = (blockIdx.y >= num_stripes);
  uint32_t column         = blockIdx.x;
  uint32_t stripe         = (is_nulldec) ? blockIdx.y - num_stripes : blockIdx.y;
  uint32_t chunk_id       = stripe * num_columns + column;
  int t                   = threadIdx.x;

  if (t < sizeof(ColumnDesc) / sizeof(uint32_t)) {
    ((volatile uint32_t *)&s->chunk)[t] = ((const uint32_t *)&chunks[chunk_id])[t];
  }
  __syncthreads();
  if (is_nulldec) {
    uint32_t null_count = 0;
    // Decode NULLs
    if (t == 0) {
      s->chunk.skip_count = 0;
      s->top.nulls.row    = 0;
      bytestream_init(&s->bs, s->chunk.streams[CI_PRESENT], s->chunk.strm_len[CI_PRESENT]);
    }
    __syncthreads();
    if (s->chunk.strm_len[CI_PRESENT] == 0) {
      // No present stream: all rows are valid
      s->vals.u32[t] = ~0;
    }
    while (s->top.nulls.row < s->chunk.num_rows) {
      uint32_t nrows_max = min(s->chunk.num_rows - s->top.nulls.row, NTHREADS * 32);
      uint32_t nrows;
      size_t row_in;

      bytestream_fill(&s->bs, t);
      __syncthreads();
      if (s->chunk.strm_len[CI_PRESENT] > 0) {
        uint32_t nbytes = Byte_RLE(&s->bs, &s->u.rle8, s->vals.u8, (nrows_max + 7) >> 3, t);
        nrows           = min(nrows_max, nbytes * 8u);
        if (!nrows) {
          // Error: mark all remaining rows as null
          nrows = nrows_max;
          if (t * 32 < nrows) { s->vals.u32[t] = 0; }
        }
      } else {
        nrows = nrows_max;
      }
      __syncthreads();
      row_in = s->chunk.start_row + s->top.nulls.row;
      if (row_in + nrows > first_row && row_in < first_row + max_num_rows &&
          s->chunk.valid_map_base != NULL) {
        int64_t dst_row   = row_in - first_row;
        int64_t dst_pos   = max(dst_row, (int64_t)0);
        uint32_t startbit = -static_cast<int32_t>(min(dst_row, (int64_t)0));
        uint32_t nbits    = nrows - min(startbit, nrows);
        uint32_t *valid   = s->chunk.valid_map_base + (dst_pos >> 5);
        uint32_t bitpos   = static_cast<uint32_t>(dst_pos) & 0x1f;
        if ((size_t)(dst_pos + nbits) > max_num_rows) {
          nbits = static_cast<uint32_t>(max_num_rows - min((size_t)dst_pos, max_num_rows));
        }
        // Store bits up to the next 32-bit aligned boundary
        if (bitpos != 0) {
          uint32_t n = min(32u - bitpos, nbits);
          if (t == 0) {
            uint32_t mask = ((1 << n) - 1) << bitpos;
            uint32_t bits = (rle8_read_bool32(s->vals.u32, startbit) << bitpos) & mask;
            atomicAnd(valid, ~mask);
            atomicOr(valid, bits);
            null_count += __popc((~bits) & mask);
          }
          nbits -= n;
          startbit += n;
          valid++;
        }
        // Store bits aligned
        if (t * 32 + 32 <= nbits) {
          uint32_t bits = rle8_read_bool32(s->vals.u32, startbit + t * 32);
          valid[t]      = bits;
          null_count += __popc(~bits);
        } else if (t * 32 < nbits) {
          uint32_t n    = nbits - t * 32;
          uint32_t mask = (1 << n) - 1;
          uint32_t bits = rle8_read_bool32(s->vals.u32, startbit + t * 32) & mask;
          atomicAnd(valid + t, ~mask);
          atomicOr(valid + t, bits);
          null_count += __popc((~bits) & mask);
        }
        __syncthreads();
      }
      // We may have some valid values that are not decoded below first_row -> count these in
      // skip_count, so that subsequent kernel can infer the correct row position
      if (row_in < first_row && t < 32) {
        uint32_t skippedrows = min(static_cast<uint32_t>(first_row - row_in), nrows);
        uint32_t skip_count  = 0;
        for (uint32_t i = t * 32; i < skippedrows; i += 32 * 32) {
          uint32_t bits = s->vals.u32[i >> 5];
          if (i + 32 > skippedrows) { bits &= (1 << (skippedrows - i)) - 1; }
          skip_count += __popc(bits);
        }
        skip_count += SHFL_XOR(skip_count, 1);
        skip_count += SHFL_XOR(skip_count, 2);
        skip_count += SHFL_XOR(skip_count, 4);
        skip_count += SHFL_XOR(skip_count, 8);
        skip_count += SHFL_XOR(skip_count, 16);
        if (t == 0) { s->chunk.skip_count += skip_count; }
      }
      __syncthreads();
      if (t == 0) { s->top.nulls.row += nrows; }
      __syncthreads();
    }
    __syncthreads();
    // Sum up the valid counts and infer null_count
    null_count += SHFL_XOR(null_count, 1);
    null_count += SHFL_XOR(null_count, 2);
    null_count += SHFL_XOR(null_count, 4);
    null_count += SHFL_XOR(null_count, 8);
    null_count += SHFL_XOR(null_count, 16);
    if (!(t & 0x1f)) { s->top.nulls.null_count[t >> 5] = null_count; }
    __syncthreads();
    if (t < 32) {
      null_count = (t < NWARPS) ? s->top.nulls.null_count[t] : 0;
      null_count += SHFL_XOR(null_count, 1);
      null_count += SHFL_XOR(null_count, 2);
      null_count += SHFL_XOR(null_count, 4);
      null_count += SHFL_XOR(null_count, 8);
      null_count += SHFL_XOR(null_count, 16);
      if (t == 0) {
        chunks[chunk_id].null_count = null_count;
        chunks[chunk_id].skip_count = s->chunk.skip_count;
      }
    }
  } else {
    // Decode string dictionary
    int encoding_kind = s->chunk.encoding_kind;
    if ((encoding_kind == DICTIONARY || encoding_kind == DICTIONARY_V2) &&
        (s->chunk.dict_len > 0)) {
      if (t == 0) {
        s->top.dict.dict_len   = s->chunk.dict_len;
        s->top.dict.local_dict = reinterpret_cast<uint2 *>(
          global_dictionary + s->chunk.dictionary_start);  // Local dictionary
        s->top.dict.dict_pos = 0;
        // CI_DATA2 contains the LENGTH stream coding the length of individual dictionary entries
        bytestream_init(&s->bs, s->chunk.streams[CI_DATA2], s->chunk.strm_len[CI_DATA2]);
      }
      __syncthreads();
      while (s->top.dict.dict_len > 0) {
        uint32_t numvals        = min(s->top.dict.dict_len, NTHREADS), len;
        volatile uint32_t *vals = s->vals.u32;
        bytestream_fill(&s->bs, t);
        __syncthreads();
        if (IS_RLEv1(s->chunk.encoding_kind)) {
          numvals = Integer_RLEv1(&s->bs, &s->u.rlev1, vals, numvals, t);
        } else  // RLEv2
        {
          numvals = Integer_RLEv2(&s->bs, &s->u.rlev2, vals, numvals, t);
        }
        __syncthreads();
        len = (t < numvals) ? vals[t] : 0;
        lengths_to_positions(vals, numvals, t);
        __syncthreads();
        if (numvals == 0) {
          // This is an error (ran out of data)
          numvals = min(s->top.dict.dict_len, NTHREADS);
          vals[t] = 0;
        }
        if (t < numvals) {
          uint2 dict_entry;
          dict_entry.x              = s->top.dict.dict_pos + vals[t] - len;
          dict_entry.y              = len;
          s->top.dict.local_dict[t] = dict_entry;
        }
        __syncthreads();
        if (t == 0) {
          s->top.dict.dict_pos += vals[numvals - 1];
          s->top.dict.dict_len -= numvals;
          s->top.dict.local_dict += numvals;
        }
        __syncthreads();
      }
    }
  }
}

/**
 * @brief Decode row positions from valid bits
 *
 * @param[in,out] s Column chunk decoder state
 * @param[in] first_row crop all rows below first rows
 * @param[in] t thread id
 *
 **/
static __device__ void DecodeRowPositions(orcdec_state_s *s, size_t first_row, int t)
{
  if (t == 0) {
    if (s->chunk.skip_count != 0) {
      s->u.rowdec.nz_count = min(min(s->chunk.skip_count, s->top.data.max_vals), NTHREADS);
      s->chunk.skip_count -= s->u.rowdec.nz_count;
      s->top.data.nrows = s->u.rowdec.nz_count;
    } else {
      s->u.rowdec.nz_count = 0;
    }
  }
  __syncthreads();
  if (t < s->u.rowdec.nz_count) {
    s->u.rowdec.row[t] = 0;  // Skipped values (below first_row)
  }
  while (s->u.rowdec.nz_count < s->top.data.max_vals &&
         s->top.data.cur_row + s->top.data.nrows < s->top.data.end_row) {
    uint32_t nrows = min(s->top.data.end_row - s->top.data.cur_row,
                         min((ROWDEC_BFRSZ - s->u.rowdec.nz_count) * 2, NTHREADS));
    if (s->chunk.strm_len[CI_PRESENT] > 0) {
      // We have a present stream
      uint32_t rmax  = s->top.data.end_row - min((uint32_t)first_row, s->top.data.end_row);
      uint32_t r     = (uint32_t)(s->top.data.cur_row + s->top.data.nrows + t - first_row);
      uint32_t valid = (t < nrows && r < rmax)
                         ? (((const uint8_t *)s->chunk.valid_map_base)[r >> 3] >> (r & 7)) & 1
                         : 0;
      volatile uint16_t *row_ofs_plus1 =
        (volatile uint16_t *)&s->u.rowdec.row[s->u.rowdec.nz_count];
      uint32_t nz_pos, row_plus1, nz_count = s->u.rowdec.nz_count, last_row;
      if (t < nrows) { row_ofs_plus1[t] = valid; }
      lengths_to_positions<uint16_t>(row_ofs_plus1, nrows, t);
      if (t < nrows) {
        nz_count += row_ofs_plus1[t];
        row_plus1 = s->top.data.nrows + t + 1;
      } else {
        row_plus1 = 0;
      }
      if (t == nrows - 1) { s->u.rowdec.nz_count = min(nz_count, s->top.data.max_vals); }
      __syncthreads();
      // TBD: Brute-forcing this, there might be a more efficient way to find the thread with the
      // last row
      last_row = (nz_count == s->u.rowdec.nz_count) ? row_plus1 : 0;
      last_row = max(last_row, SHFL_XOR(last_row, 1));
      last_row = max(last_row, SHFL_XOR(last_row, 2));
      last_row = max(last_row, SHFL_XOR(last_row, 4));
      last_row = max(last_row, SHFL_XOR(last_row, 8));
      last_row = max(last_row, SHFL_XOR(last_row, 16));
      if (!(t & 0x1f)) { *(volatile uint32_t *)&s->u.rowdec.last_row[t >> 5] = last_row; }
      nz_pos = (valid) ? nz_count : 0;
      __syncthreads();
      if (t < 32) {
        last_row = (t < NWARPS) ? *(volatile uint32_t *)&s->u.rowdec.last_row[t] : 0;
        last_row = max(last_row, SHFL_XOR(last_row, 1));
        last_row = max(last_row, SHFL_XOR(last_row, 2));
        last_row = max(last_row, SHFL_XOR(last_row, 4));
        last_row = max(last_row, SHFL_XOR(last_row, 8));
        last_row = max(last_row, SHFL_XOR(last_row, 16));
        if (t == 0) { s->top.data.nrows = last_row; }
      }
      if (valid && nz_pos - 1 < s->u.rowdec.nz_count) { s->u.rowdec.row[nz_pos - 1] = row_plus1; }
      __syncthreads();
    } else {
      // All values are valid
      nrows = min(nrows, s->top.data.max_vals - s->u.rowdec.nz_count);
      if (t < nrows) { s->u.rowdec.row[s->u.rowdec.nz_count + t] = s->top.data.nrows + t + 1; }
      __syncthreads();
      if (t == 0) {
        s->top.data.nrows += nrows;
        s->u.rowdec.nz_count += nrows;
      }
      __syncthreads();
    }
  }
}

/**
 * @brief Convert seconds from writer timezone to UTC
 *
 * @param[in] s Orc data decoder state
 * @param[in] table Timezone translation table
 * @param[in] ts Local time in seconds
 *
 * @return UTC time in seconds
 *
 **/
static __device__ int64_t ConvertToUTC(const orc_datadec_state_s *s,
                                       const int64_t *table,
                                       int64_t ts)
{
  uint32_t num_entries     = s->tz_num_entries;
  uint32_t dst_cycle       = s->tz_dst_cycle;
  int64_t first_transition = s->first_tz_transition;
  int64_t last_transition  = s->last_tz_transition;
  int64_t tsbase;
  uint32_t first, last;

  if (ts <= first_transition) {
    return ts + table[0 * 2 + 2];
  } else if (ts <= last_transition) {
    first  = 0;
    last   = num_entries - 1;
    tsbase = ts;
  } else if (!dst_cycle) {
    return ts + table[(num_entries - 1) * 2 + 2];
  } else {
    // Apply 400-year cycle rule
    const int64_t k400Years = (365 * 400 + (100 - 3)) * 24 * 60 * 60ll;
    tsbase                  = ts;
    ts %= k400Years;
    if (ts < 0) { ts += k400Years; }
    first = num_entries;
    last  = num_entries + dst_cycle - 1;
    if (ts < table[num_entries * 2 + 1]) { return tsbase + table[last * 2 + 2]; }
  }
  // Binary search the table from first to last for ts
  do {
    uint32_t mid = first + ((last - first + 1) >> 1);
    int64_t tmid = table[mid * 2 + 1];
    if (tmid <= ts) {
      first = mid;
    } else {
      if (mid == last) { break; }
      last = mid;
    }
  } while (first < last);
  return tsbase + table[first * 2 + 2];
}

/**
 * @brief Trailing zeroes for decoding timestamp nanoseconds
 *
 **/
static const __device__ __constant__ uint32_t kTimestampNanoScale[8] = {
  1, 100, 1000, 10000, 100000, 1000000, 10000000, 100000000};

/**
 * @brief Decodes column data
 *
 * @param[in] chunks ColumnDesc device array
 * @param[in] global_dictionary Global dictionary device array
 * @param[in] tz_table Timezone translation table
 * @param[in] row_groups Optional row index data
 * @param[in] max_num_rows Maximum number of rows to load
 * @param[in] first_row Crop all rows below first_row
 * @param[in] num_chunks Number of column chunks (num_columns * num_stripes)
 * @param[in] tz_len Length of timezone translation table (number of pairs)
 * @param[in] num_rowgroups Number of row groups in row index data
 * @param[in] rowidx_stride Row index stride
 *
 **/
// blockDim {NTHREADS,1,1}
extern "C" __global__ void __launch_bounds__(NTHREADS)
  gpuDecodeOrcColumnData(ColumnDesc *chunks,
                         DictionaryEntry *global_dictionary,
                         int64_t *tz_table,
                         const RowGroup *row_groups,
                         size_t max_num_rows,
                         size_t first_row,
                         uint32_t num_columns,
                         uint32_t tz_len,
                         uint32_t num_rowgroups,
                         uint32_t rowidx_stride)
{
  __shared__ __align__(16) orcdec_state_s state_g;

  orcdec_state_s *const s = &state_g;
  uint32_t chunk_id;
  int t = threadIdx.x;

  if (num_rowgroups > 0) {
    if (t < sizeof(RowGroup) / sizeof(uint32_t)) {
      ((volatile uint32_t *)&s->top.data.index)[t] =
        ((const uint32_t *)&row_groups[blockIdx.y * num_columns + blockIdx.x])[t];
    }
    __syncthreads();
    chunk_id = s->top.data.index.chunk_id;
  } else {
    chunk_id = blockIdx.x;
  }
  if (t < sizeof(ColumnDesc) / sizeof(uint32_t)) {
    ((volatile uint32_t *)&s->chunk)[t] = ((const uint32_t *)&chunks[chunk_id])[t];
  }
  __syncthreads();
  if (t == 0) {
    // If we have an index, seek to the initial run and update row positions
    if (num_rowgroups > 0) {
      uint32_t ofs0 = min(s->top.data.index.strm_offset[0], s->chunk.strm_len[CI_DATA]);
      uint32_t ofs1 = min(s->top.data.index.strm_offset[1], s->chunk.strm_len[CI_DATA2]);
      uint32_t rowgroup_rowofs;
      s->chunk.streams[CI_DATA] += ofs0;
      s->chunk.strm_len[CI_DATA] -= ofs0;
      s->chunk.streams[CI_DATA2] += ofs1;
      s->chunk.strm_len[CI_DATA2] -= ofs1;
      rowgroup_rowofs = min((blockIdx.y - min(s->chunk.rowgroup_id, blockIdx.y)) * rowidx_stride,
                            s->chunk.num_rows);
      s->chunk.start_row += rowgroup_rowofs;
      s->chunk.num_rows -= rowgroup_rowofs;
    }
    s->is_string = (s->chunk.type_kind == STRING || s->chunk.type_kind == BINARY ||
                    s->chunk.type_kind == VARCHAR || s->chunk.type_kind == CHAR);
    s->top.data.cur_row =
      max(s->chunk.start_row, max((int32_t)(first_row - s->chunk.skip_count), 0));
    s->top.data.end_row        = s->chunk.start_row + s->chunk.num_rows;
    s->top.data.buffered_count = 0;
    if (s->top.data.end_row > first_row + max_num_rows) {
      s->top.data.end_row = static_cast<uint32_t>(first_row + max_num_rows);
    }
    if (num_rowgroups > 0) {
      s->top.data.end_row = min(s->top.data.end_row, s->chunk.start_row + rowidx_stride);
    }
    if (!IS_DICTIONARY(s->chunk.encoding_kind)) { s->chunk.dictionary_start = 0; }
    if (tz_len > 0) {
      if (tz_len > 800)  // 2 entries/year for 400 years
      {
        s->top.data.tz_num_entries = tz_len - 800;
        s->top.data.tz_dst_cycle   = 800;
      } else {
        s->top.data.tz_num_entries = tz_len;
        s->top.data.tz_dst_cycle   = 0;
      }
      s->top.data.utc_epoch = kORCTimeToUTC - tz_table[0];
      if (tz_len > 0) {
        s->top.data.first_tz_transition = tz_table[1];
        s->top.data.last_tz_transition  = tz_table[(s->top.data.tz_num_entries - 1) * 2 + 1];
      }
    } else {
      s->top.data.utc_epoch = kORCTimeToUTC;
    }
    bytestream_init(&s->bs, s->chunk.streams[CI_DATA], s->chunk.strm_len[CI_DATA]);
    bytestream_init(&s->bs2, s->chunk.streams[CI_DATA2], s->chunk.strm_len[CI_DATA2]);
  }
  __syncthreads();
  while (s->top.data.cur_row < s->top.data.end_row) {
    bytestream_fill(&s->bs, t);
    bytestream_fill(&s->bs2, t);
    __syncthreads();
    if (t == 0) {
      uint32_t max_vals = s->chunk.start_row + s->chunk.num_rows - s->top.data.cur_row;
      if (num_rowgroups > 0 && (s->is_string || s->chunk.type_kind == TIMESTAMP)) {
        max_vals +=
          s->top.data.index.run_pos[IS_DICTIONARY(s->chunk.encoding_kind) ? CI_DATA : CI_DATA2];
      }
      s->bs.fill_count  = 0;
      s->bs2.fill_count = 0;
      s->top.data.nrows = 0;
      s->top.data.max_vals =
        min(max_vals, (s->chunk.type_kind == BOOLEAN) ? NTHREADS * 2 : NTHREADS);
    }
    __syncthreads();
    // Decode data streams
    {
      uint32_t numvals      = s->top.data.max_vals, secondary_val;
      uint32_t vals_skipped = 0;
      if (s->is_string || s->chunk.type_kind == TIMESTAMP) {
        // For these data types, we have a secondary unsigned 32-bit data stream
        orc_bytestream_s *bs = (IS_DICTIONARY(s->chunk.encoding_kind)) ? &s->bs : &s->bs2;
        uint32_t ofs         = 0;
        if (s->chunk.type_kind == TIMESTAMP) {
          // Restore buffered secondary stream values, if any
          ofs = s->top.data.buffered_count;
          if (ofs > 0) {
            __syncthreads();
            if (t == 0) { s->top.data.buffered_count = 0; }
          }
        }
        if (numvals > ofs) {
          if (IS_RLEv1(s->chunk.encoding_kind)) {
            numvals = ofs + Integer_RLEv1(bs, &s->u.rlev1, &s->vals.u32[ofs], numvals - ofs, t);
          } else {
            numvals = ofs + Integer_RLEv2(bs, &s->u.rlev2, &s->vals.u32[ofs], numvals - ofs, t);
          }
          __syncthreads();
          if (numvals <= ofs && t >= ofs && t < s->top.data.max_vals) { s->vals.u32[t] = 0; }
        }
        // If we're using an index, we may have to drop values from the initial run
        if (num_rowgroups > 0) {
          int cid          = IS_DICTIONARY(s->chunk.encoding_kind) ? CI_DATA : CI_DATA2;
          uint32_t run_pos = s->top.data.index.run_pos[cid];
          if (run_pos) {
            vals_skipped = min(numvals, run_pos);
            __syncthreads();
            if (t == 0) { s->top.data.index.run_pos[cid] = 0; }
            numvals -= vals_skipped;
            if (t < numvals) { secondary_val = s->vals.u32[vals_skipped + t]; }
            __syncthreads();
            if (t < numvals) { s->vals.u32[t] = secondary_val; }
          }
        }
        __syncthreads();
        // For strings with direct encoding, we need to convert the lengths into an offset
        if (!IS_DICTIONARY(s->chunk.encoding_kind)) {
          secondary_val = (t < numvals) ? s->vals.u32[t] : 0;
          if (s->chunk.type_kind != TIMESTAMP) {
            lengths_to_positions(s->vals.u32, numvals, t);
            __syncthreads();
          }
        }
        // Adjust the maximum number of values
        if (numvals == 0 && vals_skipped == 0) {
          numvals = s->top.data.max_vals;  // Just so that we don't hang if the stream is corrupted
        }
        if (t == 0 && numvals < s->top.data.max_vals) { s->top.data.max_vals = numvals; }
      }
      __syncthreads();
      // Account for skipped values
      if (num_rowgroups > 0 && !s->is_string) {
        uint32_t run_pos = s->top.data.index.run_pos[CI_DATA]
                           << ((s->chunk.type_kind == BOOLEAN) ? 3 : 0);
        numvals = min(numvals + run_pos, (s->chunk.type_kind == BOOLEAN) ? NTHREADS * 2 : NTHREADS);
      }
      // Decode the primary data stream
      if (s->chunk.type_kind == INT || s->chunk.type_kind == DATE || s->chunk.type_kind == SHORT) {
        // Signed int32 primary data stream
        if (IS_RLEv1(s->chunk.encoding_kind)) {
          numvals = Integer_RLEv1(&s->bs, &s->u.rlev1, s->vals.i32, numvals, t);
        } else {
          numvals = Integer_RLEv2(&s->bs, &s->u.rlev2, s->vals.i32, numvals, t);
        }
        __syncthreads();
      } else if (s->chunk.type_kind == BYTE) {
        numvals = Byte_RLE(&s->bs, &s->u.rle8, s->vals.u8, numvals, t);
        __syncthreads();
      } else if (s->chunk.type_kind == BOOLEAN) {
        int n = ((numvals + 7) >> 3);
        if (n > s->top.data.buffered_count) {
          numvals = Byte_RLE(&s->bs,
                             &s->u.rle8,
                             &s->vals.u8[s->top.data.buffered_count],
                             n - s->top.data.buffered_count,
                             t) +
                    s->top.data.buffered_count;
        } else {
          numvals = s->top.data.buffered_count;
        }
        __syncthreads();
        if (t == 0) {
          s->top.data.buffered_count = 0;
          s->top.data.max_vals       = min(s->top.data.max_vals, NTHREADS);
        }
        __syncthreads();
        n = numvals - ((s->top.data.max_vals + 7) >> 3);
        if (t < n) {
          secondary_val = s->vals.u8[((s->top.data.max_vals + 7) >> 3) + t];
          if (t == 0) { s->top.data.buffered_count = n; }
        }
        numvals = min(numvals << 3u, s->top.data.max_vals);
      } else if (s->chunk.type_kind == LONG || s->chunk.type_kind == TIMESTAMP ||
                 s->chunk.type_kind == DECIMAL) {
        orc_bytestream_s *bs = (s->chunk.type_kind == DECIMAL) ? &s->bs2 : &s->bs;
        if (IS_RLEv1(s->chunk.encoding_kind)) {
          numvals = Integer_RLEv1<int64_t>(bs, &s->u.rlev1, s->vals.i64, numvals, t);
        } else {
          numvals = Integer_RLEv2<int64_t>(bs, &s->u.rlev2, s->vals.i64, numvals, t);
        }
        if (s->chunk.type_kind == DECIMAL) {
          // If we're using an index, we may have to drop values from the initial run
          uint32_t skip = 0;
          int val_scale;
          if (num_rowgroups > 0) {
            uint32_t run_pos = s->top.data.index.run_pos[CI_DATA2];
            if (run_pos) {
              skip = min(numvals, run_pos);
              __syncthreads();
              if (t == 0) { s->top.data.index.run_pos[CI_DATA2] = 0; }
              numvals -= skip;
            }
          }
          val_scale = (t < numvals) ? (int)s->vals.i64[skip + t] : 0;
          __syncthreads();
          numvals = Decode_Decimals(
            &s->bs, &s->u.rle8, s->vals.i64, val_scale, numvals, s->chunk.decimal_scale, t);
        }
        __syncthreads();
      } else if (s->chunk.type_kind == FLOAT) {
        numvals = min(numvals, (BYTESTREAM_BFRSZ - 8u) >> 2);
        if (t < numvals) { s->vals.u32[t] = bytestream_readu32(&s->bs, s->bs.pos + t * 4); }
        __syncthreads();
        if (t == 0) { bytestream_flush_bytes(&s->bs, numvals * 4); }
        __syncthreads();
      } else if (s->chunk.type_kind == DOUBLE) {
        numvals = min(numvals, (BYTESTREAM_BFRSZ - 8u) >> 3);
        if (t < numvals) { s->vals.u64[t] = bytestream_readu64(&s->bs, s->bs.pos + t * 8); }
        __syncthreads();
        if (t == 0) { bytestream_flush_bytes(&s->bs, numvals * 8); }
        __syncthreads();
      }
      __syncthreads();
      if (numvals == 0 && vals_skipped != 0 && num_rowgroups > 0) {
        // Special case if the secondary streams produced fewer values than the primary stream's RLE
        // run, as a result of initial RLE run offset: keep vals_skipped as non-zero to ensure
        // proper buffered_count/max_vals update below.
      } else {
        vals_skipped = 0;
        if (num_rowgroups > 0) {
          uint32_t run_pos = s->top.data.index.run_pos[CI_DATA]
                             << ((s->chunk.type_kind == BOOLEAN) ? 3 : 0);
          if (run_pos) {
            vals_skipped = min(numvals, run_pos);
            numvals -= vals_skipped;
            __syncthreads();
            if (t == 0) { s->top.data.index.run_pos[CI_DATA] = 0; }
          }
        }
      }
      if (t == 0 && numvals + vals_skipped > 0 && numvals < s->top.data.max_vals) {
        if (s->chunk.type_kind == TIMESTAMP) {
          s->top.data.buffered_count = s->top.data.max_vals - numvals;
        }
        s->top.data.max_vals = numvals;
      }
      __syncthreads();
      // Use the valid bits to compute non-null row positions until we get a full batch of values to
      // decode
      DecodeRowPositions(s, first_row, t);
      if (!s->top.data.nrows && !s->u.rowdec.nz_count && !vals_skipped) {
        // This is a bug (could happen with bitstream errors with a bad run that would produce more
        // values than the number of remaining rows)
        return;
      }
      // Store decoded values to output
      if (t < min(min(s->top.data.max_vals, s->u.rowdec.nz_count), s->top.data.nrows) &&
          s->u.rowdec.row[t] != 0 &&
          s->top.data.cur_row + s->u.rowdec.row[t] - 1 < s->top.data.end_row) {
        size_t row = s->top.data.cur_row + s->u.rowdec.row[t] - 1 - first_row;
        if (row < max_num_rows) {
          void *data_out = s->chunk.column_data_base;
          switch (s->chunk.type_kind) {
            case FLOAT:
            case INT: static_cast<uint32_t *>(data_out)[row] = s->vals.u32[t + vals_skipped]; break;
            case DOUBLE:
            case LONG:
            case DECIMAL:
              static_cast<uint64_t *>(data_out)[row] = s->vals.u64[t + vals_skipped];
              break;
            case SHORT:
              static_cast<uint16_t *>(data_out)[row] =
                static_cast<uint16_t>(s->vals.u32[t + vals_skipped]);
              break;
            case BYTE: static_cast<uint8_t *>(data_out)[row] = s->vals.u8[t + vals_skipped]; break;
            case BOOLEAN:
              static_cast<uint8_t *>(data_out)[row] =
                (s->vals.u8[(t + vals_skipped) >> 3] >> ((~t) & 7)) & 1;
              break;
            case DATE:
              if (s->chunk.dtype_len == 8) {
                // Convert from days to milliseconds by multiplying by 24*3600*1000
                static_cast<int64_t *>(data_out)[row] =
                  86400000ll * (int64_t)s->vals.i32[t + vals_skipped];
              } else {
                static_cast<uint32_t *>(data_out)[row] = s->vals.u32[t + vals_skipped];
              }
              break;
            case STRING:
            case BINARY:
            case VARCHAR:
            case CHAR: {
              nvstrdesc_s *strdesc = &static_cast<nvstrdesc_s *>(data_out)[row];
              const uint8_t *ptr;
              uint32_t count;
              if (IS_DICTIONARY(s->chunk.encoding_kind)) {
                uint32_t dict_idx = s->vals.u32[t + vals_skipped];
                ptr               = s->chunk.streams[CI_DICTIONARY];
                if (dict_idx < s->chunk.dict_len) {
                  ptr += global_dictionary[s->chunk.dictionary_start + dict_idx].pos;
                  count = global_dictionary[s->chunk.dictionary_start + dict_idx].len;
                } else {
                  count = 0;
                  // ptr = (uint8_t *)0xdeadbeef;
                }
              } else {
                uint32_t dict_idx =
                  s->chunk.dictionary_start + s->vals.u32[t + vals_skipped] - secondary_val;
                count = secondary_val;
                ptr   = s->chunk.streams[CI_DATA] + dict_idx;
                if (dict_idx + count > s->chunk.strm_len[CI_DATA]) {
                  count = 0;
                  // ptr = (uint8_t *)0xdeadbeef;
                }
              }
              strdesc->ptr   = reinterpret_cast<const char *>(ptr);
              strdesc->count = count;
              break;
            }
            case TIMESTAMP: {
              int64_t seconds = s->vals.i64[t + vals_skipped] + s->top.data.utc_epoch;
              uint32_t nanos  = secondary_val;
              nanos           = (nanos >> 3) * kTimestampNanoScale[nanos & 7];
              if (tz_len > 0) { seconds = ConvertToUTC(&s->top.data, tz_table, seconds); }
              if (seconds < 0 && nanos != 0) { seconds -= 1; }
              if (s->chunk.ts_clock_rate)
                static_cast<int64_t *>(data_out)[row] =
                  seconds * s->chunk.ts_clock_rate +
                  (nanos + (499999999 / s->chunk.ts_clock_rate)) /
                    (1000000000 / s->chunk.ts_clock_rate);  // Output to desired clock rate
              else
                static_cast<int64_t *>(data_out)[row] = seconds * 1000000000 + nanos;
              break;
            }
          }
        }
      }
      __syncthreads();
      // Buffer secondary stream values
      if (s->chunk.type_kind == TIMESTAMP) {
        int buffer_pos = s->top.data.max_vals;
        if (t >= buffer_pos && t < buffer_pos + s->top.data.buffered_count) {
          s->vals.u32[t - buffer_pos] = secondary_val;
        }
      } else if (s->chunk.type_kind == BOOLEAN && t < s->top.data.buffered_count) {
        s->vals.u8[t] = secondary_val;
      }
    }
    __syncthreads();
    if (t == 0) {
      s->top.data.cur_row += s->top.data.nrows;
      if (s->is_string && !IS_DICTIONARY(s->chunk.encoding_kind) && s->top.data.max_vals > 0) {
        s->chunk.dictionary_start += s->vals.u32[s->top.data.max_vals - 1];
      }
    }
    __syncthreads();
  }
}

/**
 * @brief Launches kernel for decoding NULLs and building string dictionary index tables
 *
 * @param[in] chunks ColumnDesc device array [stripe][column]
 * @param[in] global_dictionary Global dictionary device array
 * @param[in] num_columns Number of columns
 * @param[in] num_stripes Number of stripes
 * @param[in] max_rows Maximum number of rows to load
 * @param[in] first_row Crop all rows below first_row
 * @param[in] stream CUDA stream to use, default 0
 *
 * @return hipSuccess if successful, a CUDA error code otherwise
 **/
hipError_t __host__ DecodeNullsAndStringDictionaries(ColumnDesc *chunks,
                                                      DictionaryEntry *global_dictionary,
                                                      uint32_t num_columns,
                                                      uint32_t num_stripes,
                                                      size_t max_num_rows,
                                                      size_t first_row,
                                                      hipStream_t stream)
{
  dim3 dim_block(NTHREADS, 1);
  dim3 dim_grid(num_columns, num_stripes * 2);  // 1024 threads per chunk
  gpuDecodeNullsAndStringDictionaries<<<dim_grid, dim_block, 0, stream>>>(
    chunks, global_dictionary, num_columns, num_stripes, max_num_rows, first_row);
  return hipSuccess;
}

/**
 * @brief Launches kernel for decoding column data
 *
 * @param[in] chunks ColumnDesc device array [stripe][column]
 * @param[in] global_dictionary Global dictionary device array
 * @param[in] num_columns Number of columns
 * @param[in] num_stripes Number of stripes
 * @param[in] max_rows Maximum number of rows to load
 * @param[in] first_row Crop all rows below first_row
 * @param[in] tz_table Timezone translation table
 * @param[in] tz_len Length of timezone translation table
 * @param[in] row_groups Optional row index data
 * @param[in] num_rowgroups Number of row groups in row index data
 * @param[in] rowidx_stride Row index stride
 * @param[in] stream CUDA stream to use, default 0
 *
 * @return hipSuccess if successful, a CUDA error code otherwise
 **/
hipError_t __host__ DecodeOrcColumnData(ColumnDesc *chunks,
                                         DictionaryEntry *global_dictionary,
                                         uint32_t num_columns,
                                         uint32_t num_stripes,
                                         size_t max_num_rows,
                                         size_t first_row,
                                         int64_t *tz_table,
                                         size_t tz_len,
                                         const RowGroup *row_groups,
                                         uint32_t num_rowgroups,
                                         uint32_t rowidx_stride,
                                         hipStream_t stream)
{
  uint32_t num_chunks = num_columns * num_stripes;
  dim3 dim_block(NTHREADS, 1);  // 1024 threads per chunk
  dim3 dim_grid((num_rowgroups > 0) ? num_columns : num_chunks,
                (num_rowgroups > 0) ? num_rowgroups : 1);
  gpuDecodeOrcColumnData<<<dim_grid, dim_block, 0, stream>>>(chunks,
                                                             global_dictionary,
                                                             tz_table,
                                                             row_groups,
                                                             max_num_rows,
                                                             first_row,
                                                             num_columns,
                                                             (uint32_t)(tz_len >> 1),
                                                             num_rowgroups,
                                                             rowidx_stride);
  return hipSuccess;
}

}  // namespace gpu
}  // namespace orc
}  // namespace io
}  // namespace cudf
