#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019-2020, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "orc_common.h"
#include "orc_gpu.h"

#include <io/utilities/block_utils.cuh>

#include <rmm/cuda_stream_view.hpp>

#include <hipcub/hipcub.hpp>

#define LOG2_BYTESTREAM_BFRSZ 13  // Must be able to handle 512x 8-byte values

#define BYTESTREAM_BFRSZ (1 << LOG2_BYTESTREAM_BFRSZ)
#define BYTESTREAM_BFRMASK32 ((BYTESTREAM_BFRSZ - 1) >> 2)
// TODO: Should be more efficient with 512 threads per block and circular queue for values
#define LOG2_NWARPS 5  // Log2 of number of warps per threadblock
#define LOG2_NTHREADS (LOG2_NWARPS + 5)
#define NWARPS (1 << LOG2_NWARPS)
#define NTHREADS (1 << LOG2_NTHREADS)
#define ROWDEC_BFRSZ \
  (NTHREADS + 128)  // Add some margin to look ahead to future rows in case there are many zeroes

#define IS_RLEv1(encoding_mode) ((encoding_mode) < DIRECT_V2)
#define IS_RLEv2(encoding_mode) ((encoding_mode) >= DIRECT_V2)
#define IS_DICTIONARY(encoding_mode) ((encoding_mode)&1)

namespace cudf {
namespace io {
namespace orc {
namespace gpu {
static __device__ __constant__ int64_t kORCTimeToUTC =
  1420070400;  // Seconds from January 1st, 1970 to January 1st, 2015

struct int128_s {
  uint64_t lo;
  int64_t hi;
};

struct orc_bytestream_s {
  const uint8_t *base;
  uint32_t pos;
  uint32_t len;
  uint32_t fill_pos;
  uint32_t fill_count;
  union {
    uint8_t u8[BYTESTREAM_BFRSZ];
    uint32_t u32[BYTESTREAM_BFRSZ >> 2];
    uint2 u64[BYTESTREAM_BFRSZ >> 3];
  } buf;
};

struct orc_rlev1_state_s {
  uint32_t num_runs;
  uint32_t num_vals;
  int32_t run_data[NWARPS * 12];  // (delta << 24) | (count << 16) | (first_val)
};

struct orc_rlev2_state_s {
  uint32_t num_runs;
  uint32_t num_vals;
  union {
    uint32_t u32[NWARPS];
    uint64_t u64[NWARPS];
  } baseval;
  uint16_t m2_pw_byte3[NWARPS];
  int64_t delta[NWARPS];
  uint16_t runs_loc[NTHREADS];
};

struct orc_byterle_state_s {
  uint32_t num_runs;
  uint32_t num_vals;
  uint32_t runs_loc[NWARPS];
  uint32_t runs_pos[NWARPS];
};

struct orc_rowdec_state_s {
  uint32_t nz_count;
  uint32_t last_row[NWARPS];
  uint32_t row[ROWDEC_BFRSZ];  // 0=skip, >0: row position relative to cur_row
};

struct orc_strdict_state_s {
  DictionaryEntry *local_dict;
  uint32_t dict_pos;
  uint32_t dict_len;
};

struct orc_nulldec_state_s {
  uint32_t row;
  uint32_t null_count[NWARPS];
};

struct orc_datadec_state_s {
  uint32_t cur_row;         // starting row of current batch
  uint32_t end_row;         // ending row of this chunk (start_row + num_rows)
  uint32_t max_vals;        // max # of non-zero values to decode in this batch
  uint32_t nrows;           // # of rows in current batch (up to NTHREADS)
  uint32_t buffered_count;  // number of buffered values in the secondary data stream
  int64_t utc_epoch;        // kORCTimeToUTC - gmtOffset
  RowGroup index;
};

struct orcdec_state_s {
  ColumnDesc chunk;
  orc_bytestream_s bs;
  orc_bytestream_s bs2;
  int is_string;
  union {
    orc_strdict_state_s dict;
    orc_nulldec_state_s nulls;
    orc_datadec_state_s data;
  } top;
  union {
    orc_rlev1_state_s rlev1;
    orc_rlev2_state_s rlev2;
    orc_byterle_state_s rle8;
    orc_rowdec_state_s rowdec;
  } u;
  union values {
    uint8_t u8[NTHREADS * 8];
    uint32_t u32[NTHREADS * 2];
    int32_t i32[NTHREADS * 2];
    uint64_t u64[NTHREADS];
    int64_t i64[NTHREADS];
    double f64[NTHREADS];
  } vals;
};

/**
 * @brief Initializes byte stream, modifying length and start position to keep the read pointer
 * 8-byte aligned.
 *
 * Assumes that the address range [start_address & ~7, (start_address + len - 1) | 7]
 * is valid.
 *
 * @param[in,out] bs Byte stream input
 * @param[in] base Pointer to raw byte stream data
 * @param[in] len Stream length in bytes
 */
static __device__ void bytestream_init(volatile orc_bytestream_s *bs,
                                       const uint8_t *base,
                                       uint32_t len)
{
  uint32_t pos   = static_cast<uint32_t>(7 & reinterpret_cast<size_t>(base));
  bs->base       = base - pos;
  bs->pos        = (len > 0) ? pos : 0;
  bs->len        = (len + pos + 7) & ~7;
  bs->fill_pos   = 0;
  bs->fill_count = min(bs->len, BYTESTREAM_BFRSZ) >> 3;
}

/**
 * @brief Increment the read position, returns number of 64-bit slots to fill
 *
 * @param[in] bs Byte stream input
 * @param[in] bytes_consumed Number of bytes that were consumed
 *
 **/
static __device__ void bytestream_flush_bytes(volatile orc_bytestream_s *bs,
                                              uint32_t bytes_consumed)
{
  uint32_t pos     = bs->pos;
  uint32_t len     = bs->len;
  uint32_t pos_new = min(pos + bytes_consumed, len);
  bs->pos          = pos_new;
  pos              = min(pos + BYTESTREAM_BFRSZ, len);
  pos_new          = min(pos_new + BYTESTREAM_BFRSZ, len);
  bs->fill_pos     = pos;
  bs->fill_count   = (pos_new >> 3) - (pos >> 3);
}

/**
 * @brief Refill the byte stream buffer
 *
 * @param[in] bs Byte stream input
 * @param[in] t thread id
 *
 **/
static __device__ void bytestream_fill(orc_bytestream_s *bs, int t)
{
  auto const count = bs->fill_count;
  if (t < count) {
    auto const pos8 = (bs->fill_pos >> 3) + t;
    memcpy(&bs->buf.u64[pos8 & ((BYTESTREAM_BFRSZ >> 3) - 1)],
           &bs->base[pos8 * sizeof(uint2)],
           sizeof(uint2));
  }
}

/**
 * @brief Read a byte from the byte stream (byte aligned)
 *
 * @param[in] bs Byte stream input
 * @param[in] pos Position in byte stream
 * @return byte
 *
 **/
inline __device__ uint8_t bytestream_readbyte(volatile orc_bytestream_s *bs, int pos)
{
  return bs->buf.u8[pos & (BYTESTREAM_BFRSZ - 1)];
}

/**
 * @brief Read 32 bits from a byte stream (little endian, byte aligned)
 *
 * @param[in] bs Byte stream input
 * @param[in] pos Position in byte stream
 * @result bits
 *
 **/
inline __device__ uint32_t bytestream_readu32(volatile orc_bytestream_s *bs, int pos)
{
  uint32_t a = bs->buf.u32[(pos & (BYTESTREAM_BFRSZ - 1)) >> 2];
  uint32_t b = bs->buf.u32[((pos + 4) & (BYTESTREAM_BFRSZ - 1)) >> 2];
  return __funnelshift_r(a, b, (pos & 3) * 8);
}

/**
 * @brief Read 64 bits from a byte stream (little endian, byte aligned)
 *
 * @param[in] bs Byte stream input
 * @param[in] pos Position in byte stream
 * @param[in] numbits number of bits
 * @return bits
 *
 **/
inline __device__ uint64_t bytestream_readu64(volatile orc_bytestream_s *bs, int pos)
{
  uint32_t a    = bs->buf.u32[(pos & (BYTESTREAM_BFRSZ - 1)) >> 2];
  uint32_t b    = bs->buf.u32[((pos + 4) & (BYTESTREAM_BFRSZ - 1)) >> 2];
  uint32_t c    = bs->buf.u32[((pos + 8) & (BYTESTREAM_BFRSZ - 1)) >> 2];
  uint32_t lo32 = __funnelshift_r(a, b, (pos & 3) * 8);
  uint32_t hi32 = __funnelshift_r(b, c, (pos & 3) * 8);
  uint64_t v    = hi32;
  v <<= 32;
  v |= lo32;
  return v;
}

/**
 * @brief Read up to 32-bits from a byte stream (big endian)
 *
 * @param[in] bs Byte stream input
 * @param[in] bitpos Position in byte stream
 * @param[in] numbits number of bits
 * @return decoded value
 *
 **/
inline __device__ uint32_t bytestream_readbits(volatile orc_bytestream_s *bs,
                                               int bitpos,
                                               uint32_t numbits)
{
  int idx    = bitpos >> 5;
  uint32_t a = __byte_perm(bs->buf.u32[(idx + 0) & BYTESTREAM_BFRMASK32], 0, 0x0123);
  uint32_t b = __byte_perm(bs->buf.u32[(idx + 1) & BYTESTREAM_BFRMASK32], 0, 0x0123);
  return __funnelshift_l(b, a, bitpos & 0x1f) >> (32 - numbits);
}

/**
 * @brief Read up to 64-bits from a byte stream (big endian)
 *
 * @param[in] bs Byte stream input
 * @param[in] bitpos Position in byte stream
 * @param[in] numbits number of bits
 * @return decoded value
 *
 **/
inline __device__ uint64_t bytestream_readbits64(volatile orc_bytestream_s *bs,
                                                 int bitpos,
                                                 uint32_t numbits)
{
  int idx       = bitpos >> 5;
  uint32_t a    = __byte_perm(bs->buf.u32[(idx + 0) & BYTESTREAM_BFRMASK32], 0, 0x0123);
  uint32_t b    = __byte_perm(bs->buf.u32[(idx + 1) & BYTESTREAM_BFRMASK32], 0, 0x0123);
  uint32_t c    = __byte_perm(bs->buf.u32[(idx + 2) & BYTESTREAM_BFRMASK32], 0, 0x0123);
  uint32_t hi32 = __funnelshift_l(b, a, bitpos & 0x1f);
  uint32_t lo32 = __funnelshift_l(c, b, bitpos & 0x1f);
  uint64_t v    = hi32;
  v <<= 32;
  v |= lo32;
  v >>= (64 - numbits);
  return v;
}

/**
 * @brief Decode a big-endian unsigned 32-bit value
 *
 * @param[in] bs Byte stream input
 * @param[in] bitpos Position in byte stream
 * @param[in] numbits number of bits
 * @param[out] result decoded value
 *
 **/
inline __device__ void bytestream_readbe(volatile orc_bytestream_s *bs,
                                         int bitpos,
                                         uint32_t numbits,
                                         uint32_t &result)
{
  result = bytestream_readbits(bs, bitpos, numbits);
}

/**
 * @brief Decode a big-endian signed 32-bit value
 *
 * @param[in] bs Byte stream input
 * @param[in] bitpos Position in byte stream
 * @param[in] numbits number of bits
 * @param[out] result decoded value
 *
 **/
inline __device__ void bytestream_readbe(volatile orc_bytestream_s *bs,
                                         int bitpos,
                                         uint32_t numbits,
                                         int32_t &result)
{
  uint32_t u = bytestream_readbits(bs, bitpos, numbits);
  result     = (int32_t)((u >> 1u) ^ -(int32_t)(u & 1));
}

/**
 * @brief Decode a big-endian unsigned 64-bit value
 *
 * @param[in] bs Byte stream input
 * @param[in] bitpos Position in byte stream
 * @param[in] numbits number of bits
 * @param[out] result decoded value
 *
 **/
inline __device__ void bytestream_readbe(volatile orc_bytestream_s *bs,
                                         int bitpos,
                                         uint32_t numbits,
                                         uint64_t &result)
{
  result = bytestream_readbits64(bs, bitpos, numbits);
}

/**
 * @brief Decode a big-endian signed 64-bit value
 *
 * @param[in] bs Byte stream input
 * @param[in] bitpos Position in byte stream
 * @param[in] numbits number of bits
 * @param[out] result decoded value
 *
 **/
inline __device__ void bytestream_readbe(volatile orc_bytestream_s *bs,
                                         int bitpos,
                                         uint32_t numbits,
                                         int64_t &result)
{
  uint64_t u = bytestream_readbits64(bs, bitpos, numbits);
  result     = (int64_t)((u >> 1u) ^ -(int64_t)(u & 1));
}

/**
 * @brief Return the length of a base-128 varint
 *
 * @param[in] bs Byte stream input
 * @param[in] pos Position in circular byte stream buffer
 * @return length of varint in bytes
 **/
template <class T>
inline __device__ uint32_t varint_length(volatile orc_bytestream_s *bs, int pos)
{
  if (bytestream_readbyte(bs, pos) > 0x7f) {
    uint32_t next32 = bytestream_readu32(bs, pos + 1);
    uint32_t zbit   = __ffs((~next32) & 0x80808080);
    if (sizeof(T) <= 4 || zbit) {
      return 1 + (zbit >> 3);  // up to 5x7 bits
    } else {
      next32 = bytestream_readu32(bs, pos + 5);
      zbit   = __ffs((~next32) & 0x80808080);
      if (zbit) {
        return 5 + (zbit >> 3);  // up to 9x7 bits
      } else if ((sizeof(T) <= 8) || (bytestream_readbyte(bs, pos + 9) <= 0x7f)) {
        return 10;  // up to 70 bits
      } else {
        uint64_t next64 = bytestream_readu64(bs, pos + 10);
        zbit            = __ffsll((~next64) & 0x8080808080808080ull);
        if (zbit) {
          return 10 + (zbit >> 3);  // Up to 18x7 bits (126)
        } else {
          return 19;  // Up to 19x7 bits (133)
        }
      }
    }
  } else {
    return 1;
  }
}

/**
 * @brief Decodes a base-128 varint
 *
 * @param[in] bs Byte stream input
 * @param[in] pos Position in circular byte stream buffer
 * @param[in] result Unpacked value
 * @return new position in byte stream buffer
 **/
template <class T>
inline __device__ int decode_base128_varint(volatile orc_bytestream_s *bs, int pos, T &result)
{
  uint32_t v = bytestream_readbyte(bs, pos++);
  if (v > 0x7f) {
    uint32_t b = bytestream_readbyte(bs, pos++);
    v          = (v & 0x7f) | (b << 7);
    if (b > 0x7f) {
      b = bytestream_readbyte(bs, pos++);
      v = (v & 0x3fff) | (b << 14);
      if (b > 0x7f) {
        b = bytestream_readbyte(bs, pos++);
        v = (v & 0x1fffff) | (b << 21);
        if (b > 0x7f) {
          b = bytestream_readbyte(bs, pos++);
          v = (v & 0x0fffffff) | (b << 28);
          if (sizeof(T) > 4) {
            uint32_t lo = v;
            uint64_t hi;
            v = b >> 4;
            if (b > 0x7f) {
              b = bytestream_readbyte(bs, pos++);
              v = (v & 7) | (b << 3);
              if (b > 0x7f) {
                b = bytestream_readbyte(bs, pos++);
                v = (v & 0x3ff) | (b << 10);
                if (b > 0x7f) {
                  b = bytestream_readbyte(bs, pos++);
                  v = (v & 0x1ffff) | (b << 17);
                  if (b > 0x7f) {
                    b = bytestream_readbyte(bs, pos++);
                    v = (v & 0xffffff) | (b << 24);
                    if (b > 0x7f) {
                      pos++;  // last bit is redundant (extra byte implies bit63 is 1)
                    }
                  }
                }
              }
            }
            hi = v;
            hi <<= 32;
            result = hi | lo;
            return pos;
          }
        }
      }
    }
  }
  result = v;
  return pos;
}

/**
 * @brief Decodes a signed int128 encoded as base-128 varint (used for decimals)
 **/
inline __device__ int128_s decode_varint128(volatile orc_bytestream_s *bs, int pos)
{
  uint32_t b        = bytestream_readbyte(bs, pos++);
  int64_t sign_mask = -(int32_t)(b & 1);
  uint64_t v        = (b >> 1) & 0x3f;
  uint32_t bitpos   = 6;
  uint64_t lo       = v;
  uint64_t hi       = 0;
  while (b > 0x7f && bitpos < 128) {
    b = bytestream_readbyte(bs, pos++);
    v |= ((uint64_t)(b & 0x7f)) << (bitpos & 0x3f);
    if (bitpos == 62) {  // 6 + 7 * 8 = 62
      lo = v;
      v  = (b & 0x7f) >> 2;  // 64 - 62
    }
    bitpos += 7;
  }
  if (bitpos >= 64) {
    hi = v;
  } else {
    lo = v;
  }
  return {(uint64_t)(lo ^ sign_mask), (int64_t)(hi ^ sign_mask)};
}

/**
 * @brief Decodes an unsigned 32-bit varint
 **/
inline __device__ int decode_varint(volatile orc_bytestream_s *bs, int pos, uint32_t &result)
{
  uint32_t u;
  pos    = decode_base128_varint<uint32_t>(bs, pos, u);
  result = u;
  return pos;
}

/**
 * @brief Decodes an unsigned 64-bit varint
 **/
inline __device__ int decode_varint(volatile orc_bytestream_s *bs, int pos, uint64_t &result)
{
  uint64_t u;
  pos    = decode_base128_varint<uint64_t>(bs, pos, u);
  result = u;
  return pos;
}

/**
 * @brief Signed version of 32-bit decode_varint
 **/
inline __device__ int decode_varint(volatile orc_bytestream_s *bs, int pos, int32_t &result)
{
  uint32_t u;
  pos    = decode_base128_varint<uint32_t>(bs, pos, u);
  result = (int32_t)((u >> 1u) ^ -(int32_t)(u & 1));
  return pos;
}

/**
 * @brief Signed version of 64-bit decode_varint
 **/
inline __device__ int decode_varint(volatile orc_bytestream_s *bs, int pos, int64_t &result)
{
  uint64_t u;
  pos    = decode_base128_varint<uint64_t>(bs, pos, u);
  result = (int64_t)((u >> 1u) ^ -(int64_t)(u & 1));
  return pos;
}

/**
 * @brief In-place conversion from lengths to positions
 *
 * @param[in] vals input values
 * @param[in] numvals number of values
 * @param[in] t thread id
 *
 * @return number of values decoded
 **/
template <class T>
inline __device__ void lengths_to_positions(volatile T *vals, uint32_t numvals, unsigned int t)
{
  for (uint32_t n = 1; n < numvals; n <<= 1) {
    __syncthreads();
    if ((t & n) && (t < numvals)) vals[t] += vals[(t & ~n) | (n - 1)];
  }
}

/**
 * @brief ORC Integer RLEv1 decoding
 *
 * @param[in] bs input byte stream
 * @param[in] rle RLE state
 * @param[in] vals buffer for output values (uint32_t, int32_t, uint64_t or int64_t)
 * @param[in] maxvals maximum number of values to decode
 * @param[in] t thread id
 *
 * @return number of values decoded
 **/
template <class T>
static __device__ uint32_t Integer_RLEv1(
  orc_bytestream_s *bs, volatile orc_rlev1_state_s *rle, volatile T *vals, uint32_t maxvals, int t)
{
  uint32_t numvals, numruns;
  if (t == 0) {
    uint32_t maxpos  = min(bs->len, bs->pos + (BYTESTREAM_BFRSZ - 8u));
    uint32_t lastpos = bs->pos;
    numvals = numruns = 0;
    // Find the length and start location of each run
    while (numvals < maxvals && numruns < NWARPS * 12) {
      uint32_t pos = lastpos;
      uint32_t n   = bytestream_readbyte(bs, pos++);
      if (n <= 0x7f) {
        // Run
        int32_t delta;
        n = n + 3;
        if (numvals + n > maxvals) break;
        delta         = bytestream_readbyte(bs, pos++);
        vals[numvals] = pos & 0xffff;
        pos += varint_length<T>(bs, pos);
        if (pos > maxpos) break;
        rle->run_data[numruns++] = (delta << 24) | (n << 16) | numvals;
        numvals += n;
      } else {
        // Literals
        uint32_t i;
        n = 0x100 - n;
        if (numvals + n > maxvals) break;
        i = 0;
        do {
          vals[numvals + i] = pos & 0xffff;
          pos += varint_length<T>(bs, pos);
        } while (++i < n);
        if (pos > maxpos) break;
        numvals += n;
      }
      lastpos = pos;
    }
    rle->num_runs = numruns;
    rle->num_vals = numvals;
    bytestream_flush_bytes(bs, lastpos - bs->pos);
  }
  __syncthreads();
  // Expand the runs
  numruns = rle->num_runs;
  if (numruns > 0) {
    int r  = t >> 5;
    int tr = t & 0x1f;
    for (uint32_t run = r; run < numruns; run += NWARPS) {
      int32_t run_data = rle->run_data[run];
      int n            = (run_data >> 16) & 0xff;
      int delta        = run_data >> 24;
      uint32_t base    = run_data & 0x3ff;
      uint32_t pos     = vals[base] & 0xffff;
      for (int i = 1 + tr; i < n; i += 32) { vals[base + i] = ((delta * i) << 16) | pos; }
    }
    __syncthreads();
  }
  numvals = rle->num_vals;
  // Decode individual 32-bit varints
  if (t < numvals) {
    int32_t pos   = vals[t];
    int32_t delta = pos >> 16;
    T v;
    decode_varint(bs, pos, v);
    vals[t] = v + delta;
  }
  __syncthreads();
  return numvals;
}

/**
 * @brief Maps the RLEv2 5-bit length code to 6-bit length
 *
 **/
static const __device__ __constant__ uint8_t kRLEv2_W[32] = {
  1,  2,  3,  4,  5,  6,  7,  8,  9,  10, 11, 12, 13, 14, 15, 16,
  17, 18, 19, 20, 21, 22, 23, 24, 26, 28, 30, 32, 40, 48, 56, 64};

/**
 * @brief Maps the RLEv2 patch size (pw + pgw) to number of bits
 *
 * Patch size (in bits) is only allowed to be from the below set. If `pw + pgw == 34` then the size
 * of the patch in the file is the smallest size in the set that can fit 34 bits i.e.
 * `ClosestFixedBitsMap[34] == 40`
 *
 * @see https://github.com/apache/orc/commit/9faf7f5147a7bc69
 */
static const __device__ __constant__ uint8_t ClosestFixedBitsMap[65] = {
  1,  1,  2,  3,  4,  5,  6,  7,  8,  9,  10, 11, 12, 13, 14, 15, 16, 17, 18, 19, 20, 21,
  22, 23, 24, 26, 26, 28, 28, 30, 30, 32, 32, 40, 40, 40, 40, 40, 40, 40, 40, 48, 48, 48,
  48, 48, 48, 48, 48, 56, 56, 56, 56, 56, 56, 56, 56, 64, 64, 64, 64, 64, 64, 64, 64};

/**
 * @brief ORC Integer RLEv2 decoding
 *
 * @param[in] bs input byte stream
 * @param[in] rle RLE state
 * @param[in] vals buffer for output values (uint32_t, int32_t, uint64_t or int64_t)
 * @param[in] maxvals maximum number of values to decode
 * @param[in] t thread id
 *
 * @return number of values decoded
 **/
template <class T>
static __device__ uint32_t Integer_RLEv2(
  orc_bytestream_s *bs, volatile orc_rlev2_state_s *rle, volatile T *vals, uint32_t maxvals, int t)
{
  uint32_t numvals, numruns;
  int r, tr;

  if (t == 0) {
    uint32_t maxpos  = min(bs->len, bs->pos + (BYTESTREAM_BFRSZ - 8u));
    uint32_t lastpos = bs->pos;
    numvals = numruns = 0;
    // Find the length and start location of each run
    while (numvals < maxvals) {
      uint32_t pos   = lastpos;
      uint32_t byte0 = bytestream_readbyte(bs, pos++);
      uint32_t n, l;
      int mode               = byte0 >> 6;
      rle->runs_loc[numruns] = numvals;
      vals[numvals]          = lastpos;
      if (mode == 0) {
        // 00lllnnn: short repeat encoding
        l = 1 + ((byte0 >> 3) & 7);  // 1 to 8 bytes
        n = 3 + (byte0 & 7);         // 3 to 10 values
      } else {
        l = kRLEv2_W[(byte0 >> 1) & 0x1f];
        n = 1 + ((byte0 & 1) << 8) + bytestream_readbyte(bs, pos++);
        if (mode == 1) {
          // 01wwwwwn.nnnnnnnn: direct encoding
          l = (l * n + 7) >> 3;
        } else if (mode == 2) {
          // 10wwwwwn.nnnnnnnn.xxxxxxxx.yyyyyyyy: patched base encoding
          uint32_t byte2      = bytestream_readbyte(bs, pos++);
          uint32_t byte3      = bytestream_readbyte(bs, pos++);
          uint32_t bw         = 1 + (byte2 >> 5);        // base value width, 1 to 8 bytes
          uint32_t pw         = kRLEv2_W[byte2 & 0x1f];  // patch width, 1 to 64 bits
          uint32_t pgw        = 1 + (byte3 >> 5);        // patch gap width, 1 to 8 bits
          uint32_t pgw_pw_len = ClosestFixedBitsMap[min(pw + pgw, 64u)];  // ceiled patch width
          uint32_t pll        = byte3 & 0x1f;                             // patch list length
          l                   = (l * n + 7) >> 3;
          l += bw;
          l += (pll * (pgw_pw_len) + 7) >> 3;
        } else {
          // 11wwwwwn.nnnnnnnn.<base>.<delta>: delta encoding
          uint32_t deltapos = varint_length<T>(bs, pos);
          deltapos += varint_length<T>(bs, pos + deltapos);
          l = (l > 1 && n > 2) ? (l * (n - 2) + 7) >> 3 : 0;
          l += deltapos;
        }
      }
      if (numvals + n > maxvals) break;
      pos += l;
      if (pos > maxpos) break;
      lastpos = pos;
      numvals += n;
      numruns++;
    }
    rle->num_vals = numvals;
    rle->num_runs = numruns;
    bytestream_flush_bytes(bs, lastpos - bs->pos);
  }
  __syncthreads();
  // Process the runs, 1 warp per run
  numruns = rle->num_runs;
  r       = t >> 5;
  tr      = t & 0x1f;
  for (uint32_t run = r; run < numruns; run += NWARPS) {
    uint32_t base, pos, w, n;
    int mode;
    if (tr == 0) {
      uint32_t byte0;
      base  = rle->runs_loc[run];
      pos   = vals[base];
      byte0 = bytestream_readbyte(bs, pos++);
      mode  = byte0 >> 6;
      if (mode == 0) {
        T baseval;
        // 00lllnnn: short repeat encoding
        w = 8 + (byte0 & 0x38);  // 8 to 64 bits
        n = 3 + (byte0 & 7);     // 3 to 10 values
        bytestream_readbe(bs, pos * 8, w, baseval);
        if (sizeof(T) <= 4) {
          rle->baseval.u32[r] = baseval;
        } else {
          rle->baseval.u64[r] = baseval;
        }
      } else {
        w = kRLEv2_W[(byte0 >> 1) & 0x1f];
        n = 1 + ((byte0 & 1) << 8) + bytestream_readbyte(bs, pos++);
        if (mode > 1) {
          if (mode == 2) {
            // Patched base
            uint32_t byte2 = bytestream_readbyte(bs, pos++);
            uint32_t byte3 = bytestream_readbyte(bs, pos++);
            uint32_t bw    = 1 + (byte2 >> 5);        // base value width, 1 to 8 bytes
            uint32_t pw    = kRLEv2_W[byte2 & 0x1f];  // patch width, 1 to 64 bits
            if (sizeof(T) <= 4) {
              uint32_t baseval, mask;
              bytestream_readbe(bs, pos * 8, bw * 8, baseval);
              mask                = (1 << (bw * 8 - 1)) - 1;
              rle->baseval.u32[r] = (baseval > mask) ? (-(int32_t)(baseval & mask)) : baseval;
            } else {
              uint64_t baseval, mask;
              bytestream_readbe(bs, pos * 8, bw * 8, baseval);
              mask = 1;
              mask <<= (bw * 8) - 1;
              mask -= 1;
              rle->baseval.u64[r] = (baseval > mask) ? (-(int64_t)(baseval & mask)) : baseval;
            }
            rle->m2_pw_byte3[r] = (pw << 8) | byte3;
            pos += bw;
          } else {
            T baseval;
            int64_t delta;
            // Delta
            pos = decode_varint(bs, pos, baseval);
            if (sizeof(T) <= 4) {
              rle->baseval.u32[r] = baseval;
            } else {
              rle->baseval.u64[r] = baseval;
            }
            pos           = decode_varint(bs, pos, delta);
            rle->delta[r] = delta;
          }
        }
      }
    }
    base = SHFL0(base);
    mode = SHFL0(mode);
    pos  = SHFL0(pos);
    n    = SHFL0(n);
    w    = SHFL0(w);
    for (uint32_t i = tr; i < n; i += 32) {
      if (sizeof(T) <= 4) {
        if (mode == 0) {
          vals[base + i] = rle->baseval.u32[r];
        } else if (mode == 1) {
          T v;
          bytestream_readbe(bs, pos * 8 + i * w, w, v);
          vals[base + i] = v;
        } else if (mode == 2) {
          uint32_t ofs   = bytestream_readbits(bs, pos * 8 + i * w, w);
          vals[base + i] = rle->baseval.u32[r] + ofs;
        } else {
          int64_t delta = rle->delta[r];
          if (w > 1 && i > 1) {
            int32_t delta_s = (delta < 0) ? -1 : 0;
            vals[base + i] =
              (bytestream_readbits(bs, pos * 8 + (i - 2) * w, w) ^ delta_s) - delta_s;
          } else {
            vals[base + i] = (i == 0) ? 0 : static_cast<uint32_t>(delta);
          }
        }
      } else {
        if (mode == 0) {
          vals[base + i] = rle->baseval.u64[r];
        } else if (mode == 1) {
          T v;
          bytestream_readbe(bs, pos * 8 + i * w, w, v);
          vals[base + i] = v;
        } else if (mode == 2) {
          uint64_t ofs   = bytestream_readbits64(bs, pos * 8 + i * w, w);
          vals[base + i] = rle->baseval.u64[r] + ofs;
        } else {
          int64_t delta = rle->delta[r], ofs;
          if (w > 1 && i > 1) {
            int64_t delta_s = (delta < 0) ? -1 : 0;
            ofs = (bytestream_readbits64(bs, pos * 8 + (i - 2) * w, w) ^ delta_s) - delta_s;
          } else {
            ofs = (i == 0) ? 0 : delta;
          }
          vals[base + i] = ofs;
        }
      }
    }
    SYNCWARP();
    // Patch values
    if (mode == 2) {
      uint32_t pw_byte3 = rle->m2_pw_byte3[r];
      uint32_t pw       = pw_byte3 >> 8;
      uint32_t pgw      = 1 + ((pw_byte3 >> 5) & 7);  // patch gap width, 1 to 8 bits
      uint32_t pll      = pw_byte3 & 0x1f;            // patch list length
      if (pll != 0) {
        uint32_t pgw_pw_len = ClosestFixedBitsMap[min(pw + pgw, 64u)];
        uint64_t patch_pos64 =
          (tr < pll) ? bytestream_readbits64(
                         bs, pos * 8 + ((n * w + 7) & ~7) + tr * (pgw_pw_len), pgw_pw_len)
                     : 0;
        uint32_t patch_pos;
        T patch = 1;
        patch <<= pw;
        patch = (patch - 1) & (T)patch_pos64;
        patch <<= w;
        patch_pos = (uint32_t)(patch_pos64 >> pw);
        for (uint32_t k = 1; k < pll; k <<= 1) {
          uint32_t tmp = SHFL(patch_pos, (tr & ~k) | (k - 1));
          patch_pos += (tr & k) ? tmp : 0;
        }
        if (tr < pll && patch_pos < n) { vals[base + patch_pos] += patch; }
      }
    }
    SYNCWARP();
    if (mode == 3) {
      T baseval;
      for (uint32_t i = 1; i < n; i <<= 1) {
        SYNCWARP();
        for (uint32_t j = tr; j < n; j += 32) {
          if (j & i) vals[base + j] += vals[base + ((j & ~i) | (i - 1))];
        }
      }
      if (sizeof(T) <= 4)
        baseval = rle->baseval.u32[r];
      else
        baseval = rle->baseval.u64[r];
      for (uint32_t j = tr; j < n; j += 32) { vals[base + j] += baseval; }
    }
  }
  __syncthreads();
  return rle->num_vals;
}

/**
 * @brief Reads 32 booleans as a packed 32-bit value
 *
 * @param[in] vals 32-bit array of values (little-endian)
 * @param[in] bitpos bit position
 *
 * @return 32-bit value
 **/
inline __device__ uint32_t rle8_read_bool32(volatile uint32_t *vals, uint32_t bitpos)
{
  uint32_t a = vals[(bitpos >> 5) + 0];
  uint32_t b = vals[(bitpos >> 5) + 1];
  a          = __byte_perm(a, 0, 0x0123);
  b          = __byte_perm(b, 0, 0x0123);
  return __brev(__funnelshift_l(b, a, bitpos));
}

/**
 * @brief ORC Byte RLE decoding
 *
 * @param[in] bs Input byte stream
 * @param[in] rle RLE state
 * @param[in] vals output buffer for decoded 8-bit values
 * @param[in] maxvals Maximum number of values to decode
 * @param[in] t thread id
 *
 * @return number of values decoded
 **/
static __device__ uint32_t Byte_RLE(orc_bytestream_s *bs,
                                    volatile orc_byterle_state_s *rle,
                                    volatile uint8_t *vals,
                                    uint32_t maxvals,
                                    int t)
{
  uint32_t numvals, numruns;
  int r, tr;
  if (t == 0) {
    uint32_t maxpos  = min(bs->len, bs->pos + (BYTESTREAM_BFRSZ - 8u));
    uint32_t lastpos = bs->pos;
    numvals = numruns = 0;
    // Find the length and start location of each run
    while (numvals < maxvals && numruns < NWARPS) {
      uint32_t pos           = lastpos, n;
      rle->runs_pos[numruns] = pos;
      rle->runs_loc[numruns] = numvals;
      n                      = bytestream_readbyte(bs, pos++);
      if (n <= 0x7f) {
        // Run
        n = n + 3;
        pos++;
      } else {
        // Literals
        n = 0x100 - n;
        pos += n;
      }
      if (pos > maxpos || numvals + n > maxvals) { break; }
      numruns++;
      numvals += n;
      lastpos = pos;
    }
    rle->num_runs = numruns;
    rle->num_vals = numvals;
    bytestream_flush_bytes(bs, lastpos - bs->pos);
  }
  __syncthreads();
  numruns = rle->num_runs;
  r       = t >> 5;
  tr      = t & 0x1f;
  for (int run = r; run < numruns; run += NWARPS) {
    uint32_t pos = rle->runs_pos[run];
    uint32_t loc = rle->runs_loc[run];
    uint32_t n   = bytestream_readbyte(bs, pos++);
    uint32_t literal_mask;
    if (n <= 0x7f) {
      literal_mask = 0;
      n += 3;
    } else {
      literal_mask = ~0;
      n            = 0x100 - n;
    }
    for (uint32_t i = tr; i < n; i += 32) {
      vals[loc + i] = bytestream_readbyte(bs, pos + (i & literal_mask));
    }
  }
  __syncthreads();
  return rle->num_vals;
}

/**
 * @brief Powers of 10
 *
 **/
static const __device__ __constant__ double kPow10[40] = {
  1.0,   1.e1,  1.e2,  1.e3,  1.e4,  1.e5,  1.e6,  1.e7,  1.e8,  1.e9,  1.e10, 1.e11, 1.e12, 1.e13,
  1.e14, 1.e15, 1.e16, 1.e17, 1.e18, 1.e19, 1.e20, 1.e21, 1.e22, 1.e23, 1.e24, 1.e25, 1.e26, 1.e27,
  1.e28, 1.e29, 1.e30, 1.e31, 1.e32, 1.e33, 1.e34, 1.e35, 1.e36, 1.e37, 1.e38, 1.e39,
};

static const __device__ __constant__ int64_t kPow5i[28] = {1,
                                                           5,
                                                           25,
                                                           125,
                                                           625,
                                                           3125,
                                                           15625,
                                                           78125,
                                                           390625,
                                                           1953125,
                                                           9765625,
                                                           48828125,
                                                           244140625,
                                                           1220703125,
                                                           6103515625ll,
                                                           30517578125ll,
                                                           152587890625ll,
                                                           762939453125ll,
                                                           3814697265625ll,
                                                           19073486328125ll,
                                                           95367431640625ll,
                                                           476837158203125ll,
                                                           2384185791015625ll,
                                                           11920928955078125ll,
                                                           59604644775390625ll,
                                                           298023223876953125ll,
                                                           1490116119384765625ll,
                                                           7450580596923828125ll};

/**
 * @brief ORC Decimal decoding (unbounded base-128 varints)
 *
 * @param[in] bs Input byte stream
 * @param[in,out] vals on input: scale from secondary stream, on output: value
 * @param[in] numvals Number of values to decode
 * @param[in] t thread id
 *
 * @return number of values decoded
 *
 **/
static __device__ int Decode_Decimals(orc_bytestream_s *bs,
                                      volatile orc_byterle_state_s *scratch,
                                      volatile orcdec_state_s::values &vals,
                                      int val_scale,
                                      int numvals,
                                      int col_scale,
                                      int t)
{
  uint32_t num_vals_read = 0;
  // Iterates till `numvals` are read or there is nothing to read once the
  // stream has reached its end, and can't read anything more.
  while (num_vals_read != numvals) {
    if (t == 0) {
      uint32_t maxpos  = min(bs->len, bs->pos + (BYTESTREAM_BFRSZ - 8u));
      uint32_t lastpos = bs->pos;
      uint32_t n;
      for (n = num_vals_read; n < numvals; n++) {
        uint32_t pos = lastpos;
        pos += varint_length<uint4>(bs, pos);
        if (pos > maxpos) break;
        vals.i64[n] = lastpos;
        lastpos     = pos;
      }
      scratch->num_vals = n;
      bytestream_flush_bytes(bs, lastpos - bs->pos);
    }
    __syncthreads();
    uint32_t num_vals_to_read = scratch->num_vals;
    if (t >= num_vals_read and t < num_vals_to_read) {
      auto const pos = static_cast<int>(vals.i64[t]);
      int128_s v     = decode_varint128(bs, pos);

      if (col_scale & ORC_DECIMAL2FLOAT64_SCALE) {
        double f      = Int128ToDouble_rn(v.lo, v.hi);
        int32_t scale = (t < numvals) ? val_scale : 0;
        if (scale >= 0)
          vals.f64[t] = f / kPow10[min(scale, 39)];
        else
          vals.f64[t] = f * kPow10[min(-scale, 39)];
      } else {
        int32_t scale = (t < numvals) ? (col_scale & ~ORC_DECIMAL2FLOAT64_SCALE) - val_scale : 0;
        if (scale >= 0) {
          scale       = min(scale, 27);
          vals.i64[t] = ((int64_t)v.lo * kPow5i[scale]) << scale;
        } else  // if (scale < 0)
        {
          bool is_negative = (v.hi < 0);
          uint64_t hi = v.hi, lo = v.lo;
          scale = min(-scale, 27);
          if (is_negative) {
            hi = (~hi) + (lo == 0);
            lo = (~lo) + 1;
          }
          lo = (lo >> (uint32_t)scale) | ((uint64_t)hi << (64 - scale));
          hi >>= (int32_t)scale;
          if (hi != 0) {
            // Use intermediate float
            lo = __double2ull_rn(Int128ToDouble_rn(lo, hi) / __ll2double_rn(kPow5i[scale]));
            hi = 0;
          } else {
            lo /= kPow5i[scale];
          }
          vals.i64[t] = (is_negative) ? -(int64_t)lo : (int64_t)lo;
        }
      }
    }
    // There is nothing to read, so break
    if (num_vals_read == num_vals_to_read) break;

    // Update number of values read (This contains values of previous iteration)
    num_vals_read = num_vals_to_read;

    // Have to wait till all threads have copied data
    __syncthreads();
    if (num_vals_read != numvals) {
      bytestream_fill(bs, t);
      __syncthreads();
      if (t == 0) {
        // Needs to be reset since bytestream has been filled
        bs->fill_count = 0;
      }
    }
    // Adding to get all threads in sync before next read
    __syncthreads();
  }
  return num_vals_read;
}

/**
 * @brief Decoding NULLs and builds string dictionary index tables
 *
 * @param[in] chunks ColumnDesc device array [stripe][column]
 * @param[in] global_dictionary Global dictionary device array
 * @param[in] num_columns Number of columns
 * @param[in] num_stripes Number of stripes
 * @param[in] max_num_rows Maximum number of rows to load
 * @param[in] first_row Crop all rows below first_row
 *
 **/
// blockDim {NTHREADS,1,1}
template <int block_size>
__global__ void __launch_bounds__(block_size)
  gpuDecodeNullsAndStringDictionaries(ColumnDesc *chunks,
                                      DictionaryEntry *global_dictionary,
                                      uint32_t num_columns,
                                      uint32_t num_stripes,
                                      size_t max_num_rows,
                                      size_t first_row)
{
  __shared__ __align__(16) orcdec_state_s state_g;
  using warp_reduce = hipcub::WarpReduce<uint32_t>;
  __shared__ typename warp_reduce::TempStorage temp_storage[block_size / 32];

  orcdec_state_s *const s = &state_g;
  bool is_nulldec         = (blockIdx.y >= num_stripes);
  uint32_t column         = blockIdx.x;
  uint32_t stripe         = (is_nulldec) ? blockIdx.y - num_stripes : blockIdx.y;
  uint32_t chunk_id       = stripe * num_columns + column;
  int t                   = threadIdx.x;

  if (t == 0) s->chunk = chunks[chunk_id];
  __syncthreads();
  if (is_nulldec) {
    uint32_t null_count = 0;
    // Decode NULLs
    if (t == 0) {
      s->chunk.skip_count = 0;
      s->top.nulls.row    = 0;
      bytestream_init(&s->bs, s->chunk.streams[CI_PRESENT], s->chunk.strm_len[CI_PRESENT]);
    }
    __syncthreads();
    if (s->chunk.strm_len[CI_PRESENT] == 0) {
      // No present stream: all rows are valid
      s->vals.u32[t] = ~0;
    }
    while (s->top.nulls.row < s->chunk.num_rows) {
      uint32_t nrows_max = min(s->chunk.num_rows - s->top.nulls.row, blockDim.x * 32);
      uint32_t nrows;
      size_t row_in;

      bytestream_fill(&s->bs, t);
      __syncthreads();
      if (s->chunk.strm_len[CI_PRESENT] > 0) {
        uint32_t nbytes = Byte_RLE(&s->bs, &s->u.rle8, s->vals.u8, (nrows_max + 7) >> 3, t);
        nrows           = min(nrows_max, nbytes * 8u);
        if (!nrows) {
          // Error: mark all remaining rows as null
          nrows = nrows_max;
          if (t * 32 < nrows) { s->vals.u32[t] = 0; }
        }
      } else {
        nrows = nrows_max;
      }
      __syncthreads();
      row_in = s->chunk.start_row + s->top.nulls.row;
      if (row_in + nrows > first_row && row_in < first_row + max_num_rows &&
          s->chunk.valid_map_base != NULL) {
        int64_t dst_row   = row_in - first_row;
        int64_t dst_pos   = max(dst_row, (int64_t)0);
        uint32_t startbit = -static_cast<int32_t>(min(dst_row, (int64_t)0));
        uint32_t nbits    = nrows - min(startbit, nrows);
        uint32_t *valid   = s->chunk.valid_map_base + (dst_pos >> 5);
        uint32_t bitpos   = static_cast<uint32_t>(dst_pos) & 0x1f;
        if ((size_t)(dst_pos + nbits) > max_num_rows) {
          nbits = static_cast<uint32_t>(max_num_rows - min((size_t)dst_pos, max_num_rows));
        }
        // Store bits up to the next 32-bit aligned boundary
        if (bitpos != 0) {
          uint32_t n = min(32u - bitpos, nbits);
          if (t == 0) {
            uint32_t mask = ((1 << n) - 1) << bitpos;
            uint32_t bits = (rle8_read_bool32(s->vals.u32, startbit) << bitpos) & mask;
            atomicAnd(valid, ~mask);
            atomicOr(valid, bits);
            null_count += __popc((~bits) & mask);
          }
          nbits -= n;
          startbit += n;
          valid++;
        }
        // Store bits aligned
        if (t * 32 + 32 <= nbits) {
          uint32_t bits = rle8_read_bool32(s->vals.u32, startbit + t * 32);
          valid[t]      = bits;
          null_count += __popc(~bits);
        } else if (t * 32 < nbits) {
          uint32_t n    = nbits - t * 32;
          uint32_t mask = (1 << n) - 1;
          uint32_t bits = rle8_read_bool32(s->vals.u32, startbit + t * 32) & mask;
          atomicAnd(valid + t, ~mask);
          atomicOr(valid + t, bits);
          null_count += __popc((~bits) & mask);
        }
        __syncthreads();
      }
      // We may have some valid values that are not decoded below first_row -> count these in
      // skip_count, so that subsequent kernel can infer the correct row position
      if (row_in < first_row && t < 32) {
        uint32_t skippedrows = min(static_cast<uint32_t>(first_row - row_in), nrows);
        uint32_t skip_count  = 0;
        for (uint32_t i = t * 32; i < skippedrows; i += 32 * 32) {
          uint32_t bits = s->vals.u32[i >> 5];
          if (i + 32 > skippedrows) { bits &= (1 << (skippedrows - i)) - 1; }
          skip_count += __popc(bits);
        }
        skip_count = warp_reduce(temp_storage[t / 32]).Sum(skip_count);
        if (t == 0) { s->chunk.skip_count += skip_count; }
      }
      __syncthreads();
      if (t == 0) { s->top.nulls.row += nrows; }
      __syncthreads();
    }
    __syncthreads();
    // Sum up the valid counts and infer null_count
    null_count = warp_reduce(temp_storage[t / 32]).Sum(null_count);
    if (!(t & 0x1f)) { s->top.nulls.null_count[t >> 5] = null_count; }
    __syncthreads();
    if (t < 32) {
      null_count = (t < NWARPS) ? s->top.nulls.null_count[t] : 0;
      null_count = warp_reduce(temp_storage[t / 32]).Sum(null_count);
      if (t == 0) {
        chunks[chunk_id].null_count = null_count;
        chunks[chunk_id].skip_count = s->chunk.skip_count;
      }
    }
  } else {
    // Decode string dictionary
    int encoding_kind = s->chunk.encoding_kind;
    if ((encoding_kind == DICTIONARY || encoding_kind == DICTIONARY_V2) &&
        (s->chunk.dict_len > 0)) {
      if (t == 0) {
        s->top.dict.dict_len   = s->chunk.dict_len;
        s->top.dict.local_dict = global_dictionary + s->chunk.dictionary_start;  // Local dictionary
        s->top.dict.dict_pos   = 0;
        // CI_DATA2 contains the LENGTH stream coding the length of individual dictionary entries
        bytestream_init(&s->bs, s->chunk.streams[CI_DATA2], s->chunk.strm_len[CI_DATA2]);
      }
      __syncthreads();
      while (s->top.dict.dict_len > 0) {
        uint32_t numvals        = min(s->top.dict.dict_len, blockDim.x), len;
        volatile uint32_t *vals = s->vals.u32;
        bytestream_fill(&s->bs, t);
        __syncthreads();
        if (IS_RLEv1(s->chunk.encoding_kind)) {
          numvals = Integer_RLEv1(&s->bs, &s->u.rlev1, vals, numvals, t);
        } else  // RLEv2
        {
          numvals = Integer_RLEv2(&s->bs, &s->u.rlev2, vals, numvals, t);
        }
        __syncthreads();
        len = (t < numvals) ? vals[t] : 0;
        lengths_to_positions(vals, numvals, t);
        __syncthreads();
        if (numvals == 0) {
          // This is an error (ran out of data)
          numvals = min(s->top.dict.dict_len, blockDim.x);
          vals[t] = 0;
        }
        if (t < numvals) {
          s->top.dict.local_dict[t] = {s->top.dict.dict_pos + vals[t] - len, len};
        }
        __syncthreads();
        if (t == 0) {
          s->top.dict.dict_pos += vals[numvals - 1];
          s->top.dict.dict_len -= numvals;
          s->top.dict.local_dict += numvals;
        }
        __syncthreads();
      }
    }
  }
}

/**
 * @brief Decode row positions from valid bits
 *
 * @param[in,out] s Column chunk decoder state
 * @param[in] first_row crop all rows below first rows
 * @param[in] t thread id
 * @param[in] temp_storage shared memory storage to performance warp reduce
 *
 **/
template <typename Storage>
static __device__ void DecodeRowPositions(orcdec_state_s *s,
                                          size_t first_row,
                                          int t,
                                          Storage &temp_storage)
{
  using warp_reduce = hipcub::WarpReduce<uint32_t>;
  if (t == 0) {
    if (s->chunk.skip_count != 0) {
      s->u.rowdec.nz_count = min(min(s->chunk.skip_count, s->top.data.max_vals), blockDim.x);
      s->chunk.skip_count -= s->u.rowdec.nz_count;
      s->top.data.nrows = s->u.rowdec.nz_count;
    } else {
      s->u.rowdec.nz_count = 0;
    }
  }
  __syncthreads();
  if (t < s->u.rowdec.nz_count) {
    s->u.rowdec.row[t] = 0;  // Skipped values (below first_row)
  }
  while (s->u.rowdec.nz_count < s->top.data.max_vals &&
         s->top.data.cur_row + s->top.data.nrows < s->top.data.end_row) {
    uint32_t nrows = min(s->top.data.end_row - (s->top.data.cur_row + s->top.data.nrows),
                         min((ROWDEC_BFRSZ - s->u.rowdec.nz_count) * 2, blockDim.x));
    if (s->chunk.strm_len[CI_PRESENT] > 0) {
      // We have a present stream
      uint32_t rmax  = s->top.data.end_row - min((uint32_t)first_row, s->top.data.end_row);
      uint32_t r     = (uint32_t)(s->top.data.cur_row + s->top.data.nrows + t - first_row);
      uint32_t valid = (t < nrows && r < rmax)
                         ? (((const uint8_t *)s->chunk.valid_map_base)[r >> 3] >> (r & 7)) & 1
                         : 0;
      volatile uint16_t *row_ofs_plus1 =
        (volatile uint16_t *)&s->u.rowdec.row[s->u.rowdec.nz_count];
      uint32_t nz_pos, row_plus1, nz_count = s->u.rowdec.nz_count, last_row;
      if (t < nrows) { row_ofs_plus1[t] = valid; }
      lengths_to_positions<uint16_t>(row_ofs_plus1, nrows, t);
      if (t < nrows) {
        nz_count += row_ofs_plus1[t];
        row_plus1 = s->top.data.nrows + t + 1;
      } else {
        row_plus1 = 0;
      }
      if (t == nrows - 1) { s->u.rowdec.nz_count = min(nz_count, s->top.data.max_vals); }
      __syncthreads();
      // TBD: Brute-forcing this, there might be a more efficient way to find the thread with the
      // last row
      last_row = (nz_count == s->u.rowdec.nz_count) ? row_plus1 : 0;
      last_row = warp_reduce(temp_storage[t / 32]).Reduce(last_row, hipcub::Max());
      if (!(t & 0x1f)) { *(volatile uint32_t *)&s->u.rowdec.last_row[t >> 5] = last_row; }
      nz_pos = (valid) ? nz_count : 0;
      __syncthreads();
      if (t < 32) {
        last_row = (t < NWARPS) ? *(volatile uint32_t *)&s->u.rowdec.last_row[t] : 0;
        last_row = warp_reduce(temp_storage[t / 32]).Reduce(last_row, hipcub::Max());
        if (t == 0) { s->top.data.nrows = last_row; }
      }
      if (valid && nz_pos - 1 < s->u.rowdec.nz_count) { s->u.rowdec.row[nz_pos - 1] = row_plus1; }
      __syncthreads();
    } else {
      // All values are valid
      nrows = min(nrows, s->top.data.max_vals - s->u.rowdec.nz_count);
      if (t < nrows) { s->u.rowdec.row[s->u.rowdec.nz_count + t] = s->top.data.nrows + t + 1; }
      __syncthreads();
      if (t == 0) {
        s->top.data.nrows += nrows;
        s->u.rowdec.nz_count += nrows;
      }
      __syncthreads();
    }
  }
}

/**
 * @brief Trailing zeroes for decoding timestamp nanoseconds
 *
 **/
static const __device__ __constant__ uint32_t kTimestampNanoScale[8] = {
  1, 100, 1000, 10000, 100000, 1000000, 10000000, 100000000};

/**
 * @brief Decodes column data
 *
 * @param[in] chunks ColumnDesc device array
 * @param[in] global_dictionary Global dictionary device array
 * @param[in] tz_table Timezone translation table
 * @param[in] row_groups Optional row index data
 * @param[in] max_num_rows Maximum number of rows to load
 * @param[in] first_row Crop all rows below first_row
 * @param[in] num_chunks Number of column chunks (num_columns * num_stripes)
 * @param[in] num_rowgroups Number of row groups in row index data
 * @param[in] rowidx_stride Row index stride
 *
 **/
// blockDim {NTHREADS,1,1}
template <int block_size>
__global__ void __launch_bounds__(block_size)
  gpuDecodeOrcColumnData(ColumnDesc *chunks,
                         DictionaryEntry *global_dictionary,
                         timezone_table_view tz_table,
                         const RowGroup *row_groups,
                         size_t max_num_rows,
                         size_t first_row,
                         uint32_t num_columns,
                         uint32_t num_rowgroups,
                         uint32_t rowidx_stride)
{
  __shared__ __align__(16) orcdec_state_s state_g;
  __shared__ typename hipcub::WarpReduce<uint32_t>::TempStorage temp_storage[block_size / 32];

  orcdec_state_s *const s = &state_g;
  uint32_t chunk_id;
  int t = threadIdx.x;

  if (num_rowgroups > 0) {
    if (t == 0) s->top.data.index = row_groups[blockIdx.y * num_columns + blockIdx.x];
    __syncthreads();
    chunk_id = s->top.data.index.chunk_id;
  } else {
    chunk_id = blockIdx.x;
  }
  if (t == 0) s->chunk = chunks[chunk_id];

  __syncthreads();
  if (t == 0) {
    // If we have an index, seek to the initial run and update row positions
    if (num_rowgroups > 0) {
      uint32_t ofs0 = min(s->top.data.index.strm_offset[0], s->chunk.strm_len[CI_DATA]);
      uint32_t ofs1 = min(s->top.data.index.strm_offset[1], s->chunk.strm_len[CI_DATA2]);
      uint32_t rowgroup_rowofs;
      s->chunk.streams[CI_DATA] += ofs0;
      s->chunk.strm_len[CI_DATA] -= ofs0;
      s->chunk.streams[CI_DATA2] += ofs1;
      s->chunk.strm_len[CI_DATA2] -= ofs1;
      rowgroup_rowofs = min((blockIdx.y - min(s->chunk.rowgroup_id, blockIdx.y)) * rowidx_stride,
                            s->chunk.num_rows);
      s->chunk.start_row += rowgroup_rowofs;
      s->chunk.num_rows -= rowgroup_rowofs;
    }
    s->is_string = (s->chunk.type_kind == STRING || s->chunk.type_kind == BINARY ||
                    s->chunk.type_kind == VARCHAR || s->chunk.type_kind == CHAR);
    s->top.data.cur_row =
      max(s->chunk.start_row, max((int32_t)(first_row - s->chunk.skip_count), 0));
    s->top.data.end_row        = s->chunk.start_row + s->chunk.num_rows;
    s->top.data.buffered_count = 0;
    if (s->top.data.end_row > first_row + max_num_rows) {
      s->top.data.end_row = static_cast<uint32_t>(first_row + max_num_rows);
    }
    if (num_rowgroups > 0) {
      s->top.data.end_row = min(s->top.data.end_row, s->chunk.start_row + rowidx_stride);
    }
    if (!IS_DICTIONARY(s->chunk.encoding_kind)) { s->chunk.dictionary_start = 0; }

    s->top.data.utc_epoch = kORCTimeToUTC - tz_table.gmt_offset;

    bytestream_init(&s->bs, s->chunk.streams[CI_DATA], s->chunk.strm_len[CI_DATA]);
    bytestream_init(&s->bs2, s->chunk.streams[CI_DATA2], s->chunk.strm_len[CI_DATA2]);
  }
  __syncthreads();
  while (s->top.data.cur_row < s->top.data.end_row) {
    bytestream_fill(&s->bs, t);
    bytestream_fill(&s->bs2, t);
    __syncthreads();
    if (t == 0) {
      uint32_t max_vals = s->chunk.start_row + s->chunk.num_rows - s->top.data.cur_row;
      if (num_rowgroups > 0 && (s->is_string || s->chunk.type_kind == TIMESTAMP)) {
        max_vals +=
          s->top.data.index.run_pos[IS_DICTIONARY(s->chunk.encoding_kind) ? CI_DATA : CI_DATA2];
      }
      s->bs.fill_count  = 0;
      s->bs2.fill_count = 0;
      s->top.data.nrows = 0;
      s->top.data.max_vals =
        min(max_vals, (s->chunk.type_kind == BOOLEAN) ? blockDim.x * 2 : blockDim.x);
    }
    __syncthreads();
    // Decode data streams
    {
      uint32_t numvals      = s->top.data.max_vals, secondary_val;
      uint32_t vals_skipped = 0;
      if (s->is_string || s->chunk.type_kind == TIMESTAMP) {
        // For these data types, we have a secondary unsigned 32-bit data stream
        orc_bytestream_s *bs = (IS_DICTIONARY(s->chunk.encoding_kind)) ? &s->bs : &s->bs2;
        uint32_t ofs         = 0;
        if (s->chunk.type_kind == TIMESTAMP) {
          // Restore buffered secondary stream values, if any
          ofs = s->top.data.buffered_count;
          if (ofs > 0) {
            __syncthreads();
            if (t == 0) { s->top.data.buffered_count = 0; }
          }
        }
        if (numvals > ofs) {
          if (IS_RLEv1(s->chunk.encoding_kind)) {
            numvals = ofs + Integer_RLEv1(bs, &s->u.rlev1, &s->vals.u32[ofs], numvals - ofs, t);
          } else {
            numvals = ofs + Integer_RLEv2(bs, &s->u.rlev2, &s->vals.u32[ofs], numvals - ofs, t);
          }
          __syncthreads();
          if (numvals <= ofs && t >= ofs && t < s->top.data.max_vals) { s->vals.u32[t] = 0; }
        }
        // If we're using an index, we may have to drop values from the initial run
        if (num_rowgroups > 0) {
          int cid          = IS_DICTIONARY(s->chunk.encoding_kind) ? CI_DATA : CI_DATA2;
          uint32_t run_pos = s->top.data.index.run_pos[cid];
          if (run_pos) {
            vals_skipped = min(numvals, run_pos);
            __syncthreads();
            if (t == 0) { s->top.data.index.run_pos[cid] = 0; }
            numvals -= vals_skipped;
            if (t < numvals) { secondary_val = s->vals.u32[vals_skipped + t]; }
            __syncthreads();
            if (t < numvals) { s->vals.u32[t] = secondary_val; }
          }
        }
        __syncthreads();
        // For strings with direct encoding, we need to convert the lengths into an offset
        if (!IS_DICTIONARY(s->chunk.encoding_kind)) {
          secondary_val = (t < numvals) ? s->vals.u32[t] : 0;
          if (s->chunk.type_kind != TIMESTAMP) {
            lengths_to_positions(s->vals.u32, numvals, t);
            __syncthreads();
          }
        }
        // Adjust the maximum number of values
        if (numvals == 0 && vals_skipped == 0) {
          numvals = s->top.data.max_vals;  // Just so that we don't hang if the stream is corrupted
        }
        if (t == 0 && numvals < s->top.data.max_vals) { s->top.data.max_vals = numvals; }
      }
      __syncthreads();
      // Account for skipped values
      if (num_rowgroups > 0 && !s->is_string) {
        uint32_t run_pos = s->top.data.index.run_pos[CI_DATA];
        numvals =
          min(numvals + run_pos, (s->chunk.type_kind == BOOLEAN) ? blockDim.x * 2 : blockDim.x);
      }
      // Decode the primary data stream
      if (s->chunk.type_kind == INT || s->chunk.type_kind == DATE || s->chunk.type_kind == SHORT) {
        // Signed int32 primary data stream
        if (IS_RLEv1(s->chunk.encoding_kind)) {
          numvals = Integer_RLEv1(&s->bs, &s->u.rlev1, s->vals.i32, numvals, t);
        } else {
          numvals = Integer_RLEv2(&s->bs, &s->u.rlev2, s->vals.i32, numvals, t);
        }
        __syncthreads();
      } else if (s->chunk.type_kind == BYTE) {
        numvals = Byte_RLE(&s->bs, &s->u.rle8, s->vals.u8, numvals, t);
        __syncthreads();
      } else if (s->chunk.type_kind == BOOLEAN) {
        int n = ((numvals + 7) >> 3);
        if (n > s->top.data.buffered_count) {
          numvals = Byte_RLE(&s->bs,
                             &s->u.rle8,
                             &s->vals.u8[s->top.data.buffered_count],
                             n - s->top.data.buffered_count,
                             t) +
                    s->top.data.buffered_count;
        } else {
          numvals = s->top.data.buffered_count;
        }
        __syncthreads();
        if (t == 0) {
          s->top.data.buffered_count = 0;
          s->top.data.max_vals       = min(s->top.data.max_vals, blockDim.x);
        }
        __syncthreads();
        // If the condition is false, then it means that s->top.data.max_vals is last set of values.
        // And as numvals is considered to be min(`max_vals+s->top.data.index.run_pos[CI_DATA]`,
        // blockDim.x*2) we have to return numvals >= s->top.data.index.run_pos[CI_DATA].
        auto const is_last_set = (s->top.data.max_vals >= s->top.data.index.run_pos[CI_DATA]);
        auto const max_vals    = (is_last_set ? s->top.data.max_vals + 7 : blockDim.x) / 8;
        n                      = numvals - max_vals;
        if (t < n) {
          secondary_val = s->vals.u8[max_vals + t];
          if (t == 0) { s->top.data.buffered_count = n; }
        }

        numvals = min(numvals * 8, is_last_set ? s->top.data.max_vals : blockDim.x);

      } else if (s->chunk.type_kind == LONG || s->chunk.type_kind == TIMESTAMP ||
                 s->chunk.type_kind == DECIMAL) {
        orc_bytestream_s *bs = (s->chunk.type_kind == DECIMAL) ? &s->bs2 : &s->bs;
        if (IS_RLEv1(s->chunk.encoding_kind)) {
          numvals = Integer_RLEv1<int64_t>(bs, &s->u.rlev1, s->vals.i64, numvals, t);
        } else {
          numvals = Integer_RLEv2<int64_t>(bs, &s->u.rlev2, s->vals.i64, numvals, t);
        }
        if (s->chunk.type_kind == DECIMAL) {
          // If we're using an index, we may have to drop values from the initial run
          uint32_t skip = 0;
          int val_scale;
          if (num_rowgroups > 0) {
            uint32_t run_pos = s->top.data.index.run_pos[CI_DATA2];
            if (run_pos) {
              skip = min(numvals, run_pos);
              __syncthreads();
              if (t == 0) { s->top.data.index.run_pos[CI_DATA2] = 0; }
              numvals -= skip;
            }
          }
          val_scale = (t < numvals) ? (int)s->vals.i64[skip + t] : 0;
          __syncthreads();
          numvals = Decode_Decimals(
            &s->bs, &s->u.rle8, s->vals, val_scale, numvals, s->chunk.decimal_scale, t);
        }
        __syncthreads();
      } else if (s->chunk.type_kind == FLOAT) {
        numvals = min(numvals, (BYTESTREAM_BFRSZ - 8u) >> 2);
        if (t < numvals) { s->vals.u32[t] = bytestream_readu32(&s->bs, s->bs.pos + t * 4); }
        __syncthreads();
        if (t == 0) { bytestream_flush_bytes(&s->bs, numvals * 4); }
        __syncthreads();
      } else if (s->chunk.type_kind == DOUBLE) {
        numvals = min(numvals, (BYTESTREAM_BFRSZ - 8u) >> 3);
        if (t < numvals) { s->vals.u64[t] = bytestream_readu64(&s->bs, s->bs.pos + t * 8); }
        __syncthreads();
        if (t == 0) { bytestream_flush_bytes(&s->bs, numvals * 8); }
        __syncthreads();
      }
      __syncthreads();
      if (numvals == 0 && vals_skipped != 0 && num_rowgroups > 0) {
        // Special case if the secondary streams produced fewer values than the primary stream's RLE
        // run, as a result of initial RLE run offset: keep vals_skipped as non-zero to ensure
        // proper buffered_count/max_vals update below.
      } else {
        vals_skipped = 0;
        if (num_rowgroups > 0) {
          uint32_t run_pos = s->top.data.index.run_pos[CI_DATA];
          if (run_pos) {
            vals_skipped = min(numvals, run_pos);
            numvals -= vals_skipped;
            __syncthreads();
            if (t == 0) { s->top.data.index.run_pos[CI_DATA] = 0; }
          }
        }
      }
      if (t == 0 && numvals + vals_skipped > 0 && numvals < s->top.data.max_vals) {
        if (s->chunk.type_kind == TIMESTAMP) {
          s->top.data.buffered_count = s->top.data.max_vals - numvals;
        }
        s->top.data.max_vals = numvals;
      }
      __syncthreads();
      // Use the valid bits to compute non-null row positions until we get a full batch of values to
      // decode
      DecodeRowPositions(s, first_row, t, temp_storage);
      if (!s->top.data.nrows && !s->u.rowdec.nz_count && !vals_skipped) {
        // This is a bug (could happen with bitstream errors with a bad run that would produce more
        // values than the number of remaining rows)
        return;
      }
      // Store decoded values to output
      if (t < min(min(s->top.data.max_vals, s->u.rowdec.nz_count), s->top.data.nrows) &&
          s->u.rowdec.row[t] != 0 &&
          s->top.data.cur_row + s->u.rowdec.row[t] - 1 < s->top.data.end_row) {
        size_t row = s->top.data.cur_row + s->u.rowdec.row[t] - 1 - first_row;
        if (row < max_num_rows) {
          void *data_out = s->chunk.column_data_base;
          switch (s->chunk.type_kind) {
            case FLOAT:
            case INT: static_cast<uint32_t *>(data_out)[row] = s->vals.u32[t + vals_skipped]; break;
            case DOUBLE:
            case LONG:
            case DECIMAL:
              static_cast<uint64_t *>(data_out)[row] = s->vals.u64[t + vals_skipped];
              break;
            case SHORT:
              static_cast<uint16_t *>(data_out)[row] =
                static_cast<uint16_t>(s->vals.u32[t + vals_skipped]);
              break;
            case BYTE: static_cast<uint8_t *>(data_out)[row] = s->vals.u8[t + vals_skipped]; break;
            case BOOLEAN:
              static_cast<uint8_t *>(data_out)[row] =
                (s->vals.u8[(t + vals_skipped) >> 3] >> ((~(t + vals_skipped)) & 7)) & 1;
              break;
            case DATE:
              if (s->chunk.dtype_len == 8) {
                // Convert from days to milliseconds by multiplying by 24*3600*1000
                static_cast<int64_t *>(data_out)[row] =
                  86400000ll * (int64_t)s->vals.i32[t + vals_skipped];
              } else {
                static_cast<uint32_t *>(data_out)[row] = s->vals.u32[t + vals_skipped];
              }
              break;
            case STRING:
            case BINARY:
            case VARCHAR:
            case CHAR: {
              nvstrdesc_s *strdesc = &static_cast<nvstrdesc_s *>(data_out)[row];
              void const *ptr      = nullptr;
              uint32_t count       = 0;
              if (IS_DICTIONARY(s->chunk.encoding_kind)) {
                auto const dict_idx = s->vals.u32[t + vals_skipped];
                if (dict_idx < s->chunk.dict_len) {
                  auto const &g_entry = global_dictionary[s->chunk.dictionary_start + dict_idx];

                  ptr   = s->chunk.streams[CI_DICTIONARY] + g_entry.pos;
                  count = g_entry.len;
                }
              } else {
                auto const dict_idx =
                  s->chunk.dictionary_start + s->vals.u32[t + vals_skipped] - secondary_val;

                if (dict_idx + count <= s->chunk.strm_len[CI_DATA]) {
                  ptr   = s->chunk.streams[CI_DATA] + dict_idx;
                  count = secondary_val;
                }
              }
              strdesc->ptr   = static_cast<char const *>(ptr);
              strdesc->count = count;
              break;
            }
            case TIMESTAMP: {
              int64_t seconds = s->vals.i64[t + vals_skipped] + s->top.data.utc_epoch;
              uint32_t nanos  = secondary_val;
              nanos           = (nanos >> 3) * kTimestampNanoScale[nanos & 7];
              if (!tz_table.ttimes.empty()) {
                seconds += get_gmt_offset(tz_table.ttimes, tz_table.offsets, seconds);
              }
              if (seconds < 0 && nanos != 0) { seconds -= 1; }
              if (s->chunk.ts_clock_rate)
                static_cast<int64_t *>(data_out)[row] =
                  seconds * s->chunk.ts_clock_rate +
                  (nanos + (499999999 / s->chunk.ts_clock_rate)) /
                    (1000000000 / s->chunk.ts_clock_rate);  // Output to desired clock rate
              else
                static_cast<int64_t *>(data_out)[row] = seconds * 1000000000 + nanos;
              break;
            }
          }
        }
      }
      __syncthreads();
      // Buffer secondary stream values
      if (s->chunk.type_kind == TIMESTAMP) {
        int buffer_pos = s->top.data.max_vals;
        if (t >= buffer_pos && t < buffer_pos + s->top.data.buffered_count) {
          s->vals.u32[t - buffer_pos] = secondary_val;
        }
      } else if (s->chunk.type_kind == BOOLEAN && t < s->top.data.buffered_count) {
        s->vals.u8[t] = secondary_val;
      }
    }
    __syncthreads();
    if (t == 0) {
      s->top.data.cur_row += s->top.data.nrows;
      if (s->is_string && !IS_DICTIONARY(s->chunk.encoding_kind) && s->top.data.max_vals > 0) {
        s->chunk.dictionary_start += s->vals.u32[s->top.data.max_vals - 1];
      }
    }
    __syncthreads();
  }
}

/**
 * @brief Launches kernel for decoding NULLs and building string dictionary index tables
 *
 * @param[in] chunks ColumnDesc device array [stripe][column]
 * @param[in] global_dictionary Global dictionary device array
 * @param[in] num_columns Number of columns
 * @param[in] num_stripes Number of stripes
 * @param[in] max_rows Maximum number of rows to load
 * @param[in] first_row Crop all rows below first_row
 * @param[in] stream CUDA stream to use, default 0
 */
void __host__ DecodeNullsAndStringDictionaries(ColumnDesc *chunks,
                                               DictionaryEntry *global_dictionary,
                                               uint32_t num_columns,
                                               uint32_t num_stripes,
                                               size_t max_num_rows,
                                               size_t first_row,
                                               rmm::cuda_stream_view stream)
{
  dim3 dim_block(NTHREADS, 1);
  dim3 dim_grid(num_columns, num_stripes * 2);  // 1024 threads per chunk
  gpuDecodeNullsAndStringDictionaries<NTHREADS><<<dim_grid, dim_block, 0, stream.value()>>>(
    chunks, global_dictionary, num_columns, num_stripes, max_num_rows, first_row);
}

/**
 * @brief Launches kernel for decoding column data
 *
 * @param[in] chunks ColumnDesc device array [stripe][column]
 * @param[in] global_dictionary Global dictionary device array
 * @param[in] num_columns Number of columns
 * @param[in] num_stripes Number of stripes
 * @param[in] max_rows Maximum number of rows to load
 * @param[in] first_row Crop all rows below first_row
 * @param[in] tz_table Timezone translation table
 * @param[in] row_groups Optional row index data
 * @param[in] num_rowgroups Number of row groups in row index data
 * @param[in] rowidx_stride Row index stride
 * @param[in] stream CUDA stream to use, default 0
 */
void __host__ DecodeOrcColumnData(ColumnDesc *chunks,
                                  DictionaryEntry *global_dictionary,
                                  uint32_t num_columns,
                                  uint32_t num_stripes,
                                  size_t max_num_rows,
                                  size_t first_row,
                                  timezone_table_view tz_table,
                                  const RowGroup *row_groups,
                                  uint32_t num_rowgroups,
                                  uint32_t rowidx_stride,
                                  rmm::cuda_stream_view stream)
{
  uint32_t num_chunks = num_columns * num_stripes;
  dim3 dim_block(NTHREADS, 1);  // 1024 threads per chunk
  dim3 dim_grid((num_rowgroups > 0) ? num_columns : num_chunks,
                (num_rowgroups > 0) ? num_rowgroups : 1);
  gpuDecodeOrcColumnData<NTHREADS><<<dim_grid, dim_block, 0, stream.value()>>>(chunks,
                                                                               global_dictionary,
                                                                               tz_table,
                                                                               row_groups,
                                                                               max_num_rows,
                                                                               first_row,
                                                                               num_columns,
                                                                               num_rowgroups,
                                                                               rowidx_stride);
}

}  // namespace gpu
}  // namespace orc
}  // namespace io
}  // namespace cudf
