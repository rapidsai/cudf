#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */
#include <io/utilities/block_utils.cuh>
#include "orc_common.h"
#include "orc_gpu.h"

// Apache ORC reader does not handle zero-length patch lists for RLEv2 mode2
// Workaround replaces zero-length patch lists by a dummy zero patch
#define ZERO_PLL_WAR 1

namespace cudf {
namespace io {
namespace orc {
namespace gpu {
#define SCRATCH_BFRSZ (512 * 4)

static __device__ __constant__ int64_t kORCTimeToUTC =
  1420070400;  // Seconds from January 1st, 1970 to January 1st, 2015

struct byterle_enc_state_s {
  uint32_t literal_run;
  uint32_t repeat_run;
  volatile uint32_t rpt_map[(512 / 32) + 1];
};

struct intrle_enc_state_s {
  uint32_t literal_run;
  uint32_t delta_run;
  uint32_t literal_mode;
  uint32_t literal_w;
  uint32_t hdr_bytes;
  uint32_t pl_bytes;
  volatile uint32_t delta_map[(512 / 32) + 1];
  volatile union {
    uint32_t u32[(512 / 32) * 2];
    uint64_t u64[(512 / 32) * 2];
  } scratch;
};

struct strdata_enc_state_s {
  uint32_t char_count;
  uint32_t lengths_red[(512 / 32)];
  const char *str_data[512];
};

struct orcenc_state_s {
  uint32_t cur_row;       // Current row in group
  uint32_t present_rows;  // # of rows in present buffer
  uint32_t present_out;   // # of rows in present buffer that have been flushed
  uint32_t nrows;         // # of rows in current batch
  uint32_t numvals;       // # of non-zero values in current batch (<=nrows)
  uint32_t numlengths;    // # of non-zero values in DATA2 batch
  uint32_t nnz;           // Running count of non-null values
  EncChunk chunk;
  uint32_t strm_pos[CI_NUM_STREAMS];
  uint8_t valid_buf[512];  // valid map bits
  union {
    byterle_enc_state_s byterle;
    intrle_enc_state_s intrle;
    strdata_enc_state_s strenc;
    StripeDictionary dict_stripe;
  } u;
  union {
    uint8_t u8[SCRATCH_BFRSZ];  // general scratch buffer
    uint32_t u32[SCRATCH_BFRSZ / 4];
  } buf;
  union {
    uint8_t u8[2048];
    uint32_t u32[1024];
    int32_t i32[1024];
    uint64_t u64[1024];
    int64_t i64[1024];
  } vals;
  union {
    uint8_t u8[2048];
    uint32_t u32[1024];
  } lengths;
};

static inline __device__ uint32_t zigzag32(int32_t v)
{
  int32_t s = (v >> 31);
  return ((v ^ s) * 2) - s;
}
static inline __device__ uint64_t zigzag64(int64_t v)
{
  int64_t s = (v < 0) ? 1 : 0;
  return ((v ^ -s) * 2) + s;
}
static inline __device__ uint32_t CountLeadingBytes32(uint32_t v) { return __clz(v) >> 3; }
static inline __device__ uint32_t CountLeadingBytes64(uint64_t v) { return __clzll(v) >> 3; }

/**
 * @brief Raw data output
 *
 * @param[in] cid stream type (strm_pos[cid] will be updated and output stored at
 *streams[cid]+strm_pos[cid])
 * @param[in] inmask input buffer position mask for circular buffers
 * @param[in] s encoder state
 * @param[in] inbuf base input buffer
 * @param[in] inpos position in input buffer
 * @param[in] count number of bytes to encode
 * @param[in] t thread id
 *
 **/
template <StreamIndexType cid, uint32_t inmask>
static __device__ void StoreBytes(
  orcenc_state_s *s, const uint8_t *inbuf, uint32_t inpos, uint32_t count, int t)
{
  uint8_t *dst = s->chunk.streams[cid] + s->strm_pos[cid];
  while (count > 0) {
    uint32_t n = min(count, 512);
    if (t < n) { dst[t] = inbuf[(inpos + t) & inmask]; }
    dst += n;
    inpos += n;
    count -= n;
  }
  __syncthreads();
  if (!t) { s->strm_pos[cid] = static_cast<uint32_t>(dst - s->chunk.streams[cid]); }
}

/**
 * @brief ByteRLE encoder
 *
 * @param[in] cid stream type (strm_pos[cid] will be updated and output stored at
 *streams[cid]+strm_pos[cid])
 * @param[in] s encoder state
 * @param[in] inbuf base input buffer
 * @param[in] inpos position in input buffer
 * @param[in] inmask input buffer position mask for circular buffers
 * @param[in] numvals max number of values to encode
 * @param[in] flush encode all remaining values if nonzero
 * @param[in] t thread id
 *
 * @return number of input values encoded
 *
 **/
template <StreamIndexType cid, uint32_t inmask>
static __device__ uint32_t ByteRLE(
  orcenc_state_s *s, const uint8_t *inbuf, uint32_t inpos, uint32_t numvals, uint32_t flush, int t)
{
  uint8_t *dst     = s->chunk.streams[cid] + s->strm_pos[cid];
  uint32_t out_cnt = 0;

  while (numvals > 0) {
    uint8_t v0       = (t < numvals) ? inbuf[(inpos + t) & inmask] : 0;
    uint8_t v1       = (t + 1 < numvals) ? inbuf[(inpos + t + 1) & inmask] : 0;
    uint32_t rpt_map = BALLOT(t + 1 < numvals && v0 == v1), literal_run, repeat_run,
             maxvals = min(numvals, 512);
    if (!(t & 0x1f)) s->u.byterle.rpt_map[t >> 5] = rpt_map;
    __syncthreads();
    if (t == 0) {
      // Find the start of an identical 3-byte sequence
      // TBD: The two loops below could be eliminated using more ballot+ffs using warp0
      literal_run = 0;
      repeat_run  = 0;
      while (literal_run < maxvals) {
        uint32_t next = s->u.byterle.rpt_map[(literal_run >> 5) + 1];
        uint32_t mask = rpt_map & __funnelshift_r(rpt_map, next, 1);
        if (mask) {
          uint32_t literal_run_ofs = __ffs(mask) - 1;
          literal_run += literal_run_ofs;
          repeat_run = __ffs(~((rpt_map >> literal_run_ofs) >> 1));
          if (repeat_run + literal_run_ofs == 32) {
            while (next == ~0) {
              uint32_t next_idx = ((literal_run + repeat_run) >> 5) + 1;
              next              = (next_idx < 512 / 32) ? s->u.byterle.rpt_map[next_idx] : 0;
              repeat_run += 32;
            }
            repeat_run += __ffs(~next) - 1;
          }
          repeat_run = min(repeat_run + 1, maxvals - min(literal_run, maxvals));
          if (repeat_run < 3) {
            literal_run += (flush && literal_run + repeat_run >= numvals) ? repeat_run : 0;
            repeat_run = 0;
          }
          break;
        }
        rpt_map = next;
        literal_run += 32;
      }
      if (repeat_run >= 130) {
        // Limit large runs to multiples of 130
        repeat_run = (repeat_run >= 3 * 130) ? 3 * 130 : (repeat_run >= 2 * 130) ? 2 * 130 : 130;
      } else if (literal_run && literal_run + repeat_run == maxvals) {
        repeat_run = 0;  // Try again at next iteration
      }
      s->u.byterle.repeat_run  = repeat_run;
      s->u.byterle.literal_run = min(literal_run, maxvals);
    }
    __syncthreads();
    literal_run = s->u.byterle.literal_run;
    if (!flush && literal_run == numvals) {
      literal_run &= ~0x7f;
      if (!literal_run) break;
    }
    if (literal_run > 0) {
      uint32_t num_runs = (literal_run + 0x7f) >> 7;
      if (t < literal_run) {
        uint32_t run_id = t >> 7;
        uint32_t run    = min(literal_run - run_id * 128, 128);
        if (!(t & 0x7f)) dst[run_id + t] = 0x100 - run;
        dst[run_id + t + 1] = (cid == CI_PRESENT) ? __brev(v0) >> 24 : v0;
      }
      dst += num_runs + literal_run;
      out_cnt += literal_run;
      numvals -= literal_run;
      inpos += literal_run;
    }
    repeat_run = s->u.byterle.repeat_run;
    if (repeat_run > 0) {
      while (repeat_run >= 130) {
        if (t == literal_run)  // repeat_run follows literal_run
        {
          dst[0] = 0x7f;
          dst[1] = (cid == CI_PRESENT) ? __brev(v0) >> 24 : v0;
        }
        dst += 2;
        out_cnt += 130;
        numvals -= 130;
        inpos += 130;
        repeat_run -= 130;
      }
      if (!flush && repeat_run == numvals) {
        // Wait for more data in case we can continue the run later
        break;
      }
      if (repeat_run >= 3) {
        if (t == literal_run)  // repeat_run follows literal_run
        {
          dst[0] = repeat_run - 3;
          dst[1] = (cid == CI_PRESENT) ? __brev(v0) >> 24 : v0;
        }
        dst += 2;
        out_cnt += repeat_run;
        numvals -= repeat_run;
        inpos += repeat_run;
      }
    }
  }
  if (!t) { s->strm_pos[cid] = static_cast<uint32_t>(dst - s->chunk.streams[cid]); }
  __syncthreads();
  return out_cnt;
}

/**
 * @brief Maps the symbol size in bytes to RLEv2 5-bit length code
 **/
static const __device__ __constant__ uint8_t kByteLengthToRLEv2_W[9] = {
  0, 7, 15, 23, 27, 28, 29, 30, 31};

/**
 * @brief Encode a varint value, return the number of bytes written
 **/
static inline __device__ uint32_t StoreVarint(uint8_t *dst, uint64_t v)
{
  uint32_t bytecnt = 0;
  for (;;) {
    uint32_t c = (uint32_t)(v & 0x7f);
    v >>= 7u;
    if (v == 0) {
      dst[bytecnt++] = c;
      break;
    } else {
      dst[bytecnt++] = c + 0x80;
    }
  }
  return bytecnt;
}

static inline __device__ void intrle_minmax(int64_t &vmin, int64_t &vmax)
{
  vmin = INT64_MIN;
  vmax = INT64_MAX;
}
// static inline __device__ void intrle_minmax(uint64_t &vmin, uint64_t &vmax) { vmin = UINT64_C(0);
// vmax = UINT64_MAX; }
static inline __device__ void intrle_minmax(int32_t &vmin, int32_t &vmax)
{
  vmin = INT32_MIN;
  vmax = INT32_MAX;
}
static inline __device__ void intrle_minmax(uint32_t &vmin, uint32_t &vmax)
{
  vmin = UINT32_C(0);
  vmax = UINT32_MAX;
}

template <class T>
static inline __device__ void StoreBytesBigEndian(uint8_t *dst, T v, uint32_t w)
{
  for (uint32_t i = 0, b = w * 8; i < w; ++i) {
    b -= 8;
    dst[i] = static_cast<uint8_t>(v >> b);
  }
}

// Combine and store bits for symbol widths less than 8
static inline __device__ void StoreBitsBigEndian(
  uint8_t *dst, uint32_t v, uint32_t w, int num_vals, int t)
{
  if (t <= (num_vals | 0x1f)) {
    uint32_t mask;
    if (w <= 1) {
      v    = (v << 1) | (SHFL_XOR(v, 1) & 0x1);
      v    = (v << 2) | (SHFL_XOR(v, 2) & 0x3);
      v    = (v << 4) | (SHFL_XOR(v, 4) & 0xf);
      mask = 0x7;
    } else if (w <= 2) {
      v    = (v << 2) | (SHFL_XOR(v, 1) & 0x3);
      v    = (v << 4) | (SHFL_XOR(v, 2) & 0xf);
      mask = 0x3;
    } else  // if (w <= 4)
    {
      v    = (v << 4) | (SHFL_XOR(v, 1) & 0xf);
      mask = 0x1;
    }
    if (t < num_vals && !(t & mask)) { dst[(t * w) >> 3] = static_cast<uint8_t>(v); }
  }
}

/**
 * @brief Integer RLEv2 encoder
 *
 * @param[in] cid stream type (strm_pos[cid] will be updated and output stored at
 *streams[cid]+strm_pos[cid])
 * @param[in] s encoder state
 * @param[in] inbuf base input buffer
 * @param[in] inpos position in input buffer
 * @param[in] inmask input buffer position mask for circular buffers
 * @param[in] numvals max number of values to encode
 * @param[in] flush encode all remaining values if nonzero
 * @param[in] t thread id
 *
 * @return number of input values encoded
 *
 **/
template <StreamIndexType cid, class T, bool is_signed, uint32_t inmask>
static __device__ uint32_t IntegerRLE(
  orcenc_state_s *s, const T *inbuf, uint32_t inpos, uint32_t numvals, uint32_t flush, int t)
{
  uint8_t *dst     = s->chunk.streams[cid] + s->strm_pos[cid];
  uint32_t out_cnt = 0;

  while (numvals > 0) {
    T v0               = (t < numvals) ? inbuf[(inpos + t) & inmask] : 0;
    T v1               = (t + 1 < numvals) ? inbuf[(inpos + t + 1) & inmask] : 0;
    T v2               = (t + 2 < numvals) ? inbuf[(inpos + t + 2) & inmask] : 0;
    uint32_t delta_map = BALLOT(t + 2 < numvals && v1 - v0 == v2 - v1), maxvals = min(numvals, 512),
             literal_run, delta_run;
    if (!(t & 0x1f)) s->u.intrle.delta_map[t >> 5] = delta_map;
    __syncthreads();
    if (!t) {
      // Find the start of the next delta run (2 consecutive values with the same delta)
      literal_run = delta_run = 0;
      while (literal_run < maxvals) {
        if (delta_map != 0) {
          uint32_t literal_run_ofs = __ffs(delta_map) - 1;
          literal_run += literal_run_ofs;
          delta_run = __ffs(~((delta_map >> literal_run_ofs) >> 1));
          if (literal_run_ofs + delta_run == 32) {
            for (;;) {
              uint32_t delta_idx = (literal_run + delta_run) >> 5;
              delta_map          = (delta_idx < 512 / 32) ? s->u.intrle.delta_map[delta_idx] : 0;
              if (delta_map != ~0) break;
              delta_run += 32;
            }
            delta_run += __ffs(~delta_map) - 1;
          }
          delta_run += 2;
          break;
        }
        literal_run += 32;
        delta_map = s->u.intrle.delta_map[(literal_run >> 5)];
      }
      literal_run             = min(literal_run, maxvals);
      s->u.intrle.literal_run = literal_run;
      s->u.intrle.delta_run   = min(delta_run, maxvals - literal_run);
    }
    __syncthreads();
    literal_run = s->u.intrle.literal_run;
    // Find minimum and maximum values
    if (literal_run > 0) {
      // Find min & max
      T vmin, vmax;
      uint32_t literal_mode, literal_w;
      if (t < literal_run) {
        vmin = vmax = v0;
      } else {
        intrle_minmax(vmax, vmin);
      }
      vmin = min(vmin, (T)SHFL_XOR(vmin, 1));
      vmin = min(vmin, (T)SHFL_XOR(vmin, 2));
      vmin = min(vmin, (T)SHFL_XOR(vmin, 4));
      vmin = min(vmin, (T)SHFL_XOR(vmin, 8));
      vmin = min(vmin, (T)SHFL_XOR(vmin, 16));
      vmax = max(vmax, (T)SHFL_XOR(vmax, 1));
      vmax = max(vmax, (T)SHFL_XOR(vmax, 2));
      vmax = max(vmax, (T)SHFL_XOR(vmax, 4));
      vmax = max(vmax, (T)SHFL_XOR(vmax, 8));
      vmax = max(vmax, (T)SHFL_XOR(vmax, 16));
      if (!(t & 0x1f)) {
        s->u.intrle.scratch.u64[(t >> 5) * 2 + 0] = vmin;
        s->u.intrle.scratch.u64[(t >> 5) * 2 + 1] = vmax;
      }
      __syncthreads();
      if (t < 32) {
        vmin = (T)s->u.intrle.scratch.u64[(t & 0xf) * 2 + 0];
        vmax = (T)s->u.intrle.scratch.u64[(t & 0xf) * 2 + 1];
        vmin = min(vmin, (T)SHFL_XOR(vmin, 1));
        vmin = min(vmin, (T)SHFL_XOR(vmin, 2));
        vmin = min(vmin, (T)SHFL_XOR(vmin, 4));
        vmin = min(vmin, (T)SHFL_XOR(vmin, 8));
        vmax = max(vmax, (T)SHFL_XOR(vmax, 1));
        vmax = max(vmax, (T)SHFL_XOR(vmax, 2));
        vmax = max(vmax, (T)SHFL_XOR(vmax, 4));
        vmax = max(vmax, (T)SHFL_XOR(vmax, 8));
        if (t == 0) {
          uint32_t mode1_w, mode2_w;
          T vrange_mode1, vrange_mode2;
          s->u.intrle.scratch.u64[0] = (uint64_t)vmin;
          if (sizeof(T) > 4) {
            vrange_mode1 = (is_signed) ? max(zigzag64(vmin), zigzag64(vmax)) : vmax;
            vrange_mode2 = vmax - vmin;
            mode1_w      = 8 - min(CountLeadingBytes64(vrange_mode1), 7);
            mode2_w      = 8 - min(CountLeadingBytes64(vrange_mode2), 7);
          } else {
            vrange_mode1 = (is_signed) ? max(zigzag32(vmin), zigzag32(vmax)) : vmax;
            vrange_mode2 = vmax - vmin;
            mode1_w      = 4 - min(CountLeadingBytes32(vrange_mode1), 3);
            mode2_w      = 4 - min(CountLeadingBytes32(vrange_mode2), 3);
          }
          // Decide between mode1 & mode2 (also mode3 for length=2 repeat)
          if (vrange_mode2 == 0 && mode1_w > 1) {
            // Should only occur if literal_run==2 (otherwise would have resulted in repeat_run >=
            // 3)
            uint32_t bytecnt = 2;
            dst[0]           = 0xC0 + ((literal_run - 1) >> 8);
            dst[1]           = (literal_run - 1) & 0xff;
            bytecnt += StoreVarint(dst + 2, vrange_mode1);
            dst[bytecnt++]           = 0;  // Zero delta
            s->u.intrle.literal_mode = 3;
            s->u.intrle.literal_w    = bytecnt;
          } else {
            uint32_t range, w;
            if (mode1_w > mode2_w && (literal_run - 1) * (mode1_w - mode2_w) > 4) {
              s->u.intrle.literal_mode = 2;
              w                        = mode2_w;
              range                    = (uint32_t)vrange_mode2;
            } else {
              s->u.intrle.literal_mode = 1;
              w                        = mode1_w;
              range                    = (uint32_t)vrange_mode1;
            }
            if (w == 1)
              w = (range >= 16) ? w << 3 : (range >= 4) ? 4 : (range >= 2) ? 2 : 1;
            else
              w <<= 3;  // bytes -> bits
            s->u.intrle.literal_w = w;
          }
        }
      }
      __syncthreads();
      vmin         = (T)s->u.intrle.scratch.u64[0];
      literal_mode = s->u.intrle.literal_mode;
      literal_w    = s->u.intrle.literal_w;
      if (literal_mode == 1) {
        // Direct mode
        if (!t) {
          dst[0] = 0x40 +
                   ((literal_w < 8) ? literal_w - 1 : kByteLengthToRLEv2_W[literal_w >> 3]) * 2 +
                   ((literal_run - 1) >> 8);
          dst[1] = (literal_run - 1) & 0xff;
        }
        dst += 2;
        if (t < literal_run && is_signed) {
          if (sizeof(T) > 4)
            v0 = zigzag64(v0);
          else
            v0 = zigzag32(v0);
        }
        if (literal_w < 8)
          StoreBitsBigEndian(dst, (uint32_t)v0, literal_w, literal_run, t);
        else if (t < literal_run)
          StoreBytesBigEndian(dst + t * (literal_w >> 3), v0, (literal_w >> 3));
      } else if (literal_mode == 2) {
        // Patched base mode
        if (!t) {
          uint32_t bw, pw = 1, pll, pgw = 1, bv_scale = (is_signed) ? 0 : 1;
          vmax = (is_signed) ? ((vmin < 0) ? -vmin : vmin) * 2 : vmin;
          bw   = (sizeof(T) > 4) ? (8 - min(CountLeadingBytes64(vmax << bv_scale), 7))
                               : (4 - min(CountLeadingBytes32(vmax << bv_scale), 3));
#if ZERO_PLL_WAR
          // Insert a dummy zero patch
          pll                                                    = 1;
          dst[4 + bw + ((literal_run * literal_w + 7) >> 3) + 0] = 0;
          dst[4 + bw + ((literal_run * literal_w + 7) >> 3) + 1] = 0;
#else
          pll = 0;
#endif
          dst[0] = 0x80 +
                   ((literal_w < 8) ? literal_w - 1 : kByteLengthToRLEv2_W[literal_w >> 3]) * 2 +
                   ((literal_run - 1) >> 8);
          dst[1] = (literal_run - 1) & 0xff;
          dst[2] = ((bw - 1) << 5) | kByteLengthToRLEv2_W[pw];
          dst[3] = ((pgw - 1) << 5) | pll;
          if (is_signed) {
            vmax >>= 1;
            vmax |= vmin & ((T)1 << (bw * 8 - 1));
          }
          StoreBytesBigEndian(dst + 4, vmax, bw);
          s->u.intrle.hdr_bytes = 4 + bw;
          s->u.intrle.pl_bytes  = (pll * (pw * 8 + pgw) + 7) >> 3;
        }
        __syncthreads();
        dst += s->u.intrle.hdr_bytes;
        v0 -= (t < literal_run) ? vmin : 0;
        if (literal_w < 8)
          StoreBitsBigEndian(dst, (uint32_t)v0, literal_w, literal_run, t);
        else if (t < literal_run)
          StoreBytesBigEndian(dst + t * (literal_w >> 3), v0, (literal_w >> 3));
        dst += s->u.intrle.pl_bytes;
      } else {
        // Delta mode
        dst += literal_w;
        literal_w = 0;
      }
      dst += (literal_run * literal_w + 7) >> 3;
      numvals -= literal_run;
      inpos += literal_run;
      out_cnt += literal_run;
      __syncthreads();
    }
    delta_run = s->u.intrle.delta_run;
    if (delta_run > 0) {
      if (t == literal_run) {
        int64_t delta       = (int64_t)v1 - (int64_t)v0;
        uint64_t delta_base = (is_signed) ? (sizeof(T) > 4) ? zigzag64(v0) : zigzag32(v0) : v0;
        if (delta == 0 && delta_run >= 3 && delta_run <= 10) {
          // Short repeat
          uint32_t delta_bw = 8 - min(CountLeadingBytes64(delta_base), 7);
          dst[0]            = ((delta_bw - 1) << 3) + (delta_run - 3);
          for (uint32_t i = 0, b = delta_bw * 8; i < delta_bw; i++) {
            b -= 8;
            dst[1 + i] = static_cast<uint8_t>(delta_base >> b);
          }
          s->u.intrle.hdr_bytes = 1 + delta_bw;
        } else {
          // Delta
          uint64_t delta_u = zigzag64(delta);
          uint32_t bytecnt = 2;
          dst[0]           = 0xC0 + ((delta_run - 1) >> 8);
          dst[1]           = (delta_run - 1) & 0xff;
          bytecnt += StoreVarint(dst + bytecnt, delta_base);
          bytecnt += StoreVarint(dst + bytecnt, delta_u);
          s->u.intrle.hdr_bytes = bytecnt;
        }
      }
      __syncthreads();
      dst += s->u.intrle.hdr_bytes;
      numvals -= delta_run;
      inpos += delta_run;
      out_cnt += delta_run;
    }
  }
  if (!t) { s->strm_pos[cid] = static_cast<uint32_t>(dst - s->chunk.streams[cid]); }
  __syncthreads();
  return out_cnt;
}

/**
 * @brief Store a group of strings as a single concatenated string
 *
 * @param[in] dst destination buffer
 * @param[in] strenc string encoder state
 * @param[in] len(t) string length (per thread)
 * @param[in] t thread id
 *
 **/
static __device__ void StoreStringData(uint8_t *dst,
                                       strdata_enc_state_s *strenc,
                                       uint32_t len,
                                       int t)
{
  // Start with summing up all the lengths
  uint32_t pos = len;
  uint32_t wt  = t & 0x1f;
  for (uint32_t n = 1; n < 32; n <<= 1) {
    uint32_t tmp = SHFL(pos, (wt & ~n) | (n - 1));
    pos += (wt & n) ? tmp : 0;
  }
  if (wt == 0x1f) { strenc->lengths_red[t >> 5] = pos; }
  dst += pos - len;
  __syncthreads();
  if (t < 32) {
    uint32_t wlen = (wt < 16) ? strenc->lengths_red[wt] : 0;
    uint32_t wpos = wlen;
    for (uint32_t n = 1; n < 16; n <<= 1) {
      uint32_t tmp = SHFL(wpos, (wt & ~n) | (n - 1));
      wpos += (wt & n) ? tmp : 0;
    }
    if (wt < 16) { strenc->lengths_red[wt] = wpos - wlen; }
    if (wt == 0xf) {
      strenc->char_count = wpos;  // Update stream position
    }
  }
  __syncthreads();
  // TBD: Might be more efficient to loop over 4 strings and copy 8 consecutive character at a time
  // rather than have each thread to a memcpy
  if (len > 0) { memcpy(dst + strenc->lengths_red[t >> 5], strenc->str_data[t], len); }
}

/**
 * @brief In-place conversion from lengths to positions
 *
 * @param[in] vals input values
 * @param[in] numvals number of values
 * @param[in] t thread id
 *
 **/
template <class T>
inline __device__ void lengths_to_positions(volatile T *vals, uint32_t numvals, unsigned int t)
{
  for (uint32_t n = 1; n < numvals; n <<= 1) {
    __syncthreads();
    if ((t & n) && (t < numvals)) vals[t] += vals[(t & ~n) | (n - 1)];
  }
}

/**
 * @brief Timestamp scale table (powers of 10)
 **/
static const __device__ __constant__ int32_t kTimeScale[10] = {
  1000000000, 100000000, 10000000, 1000000, 100000, 10000, 1000, 100, 10, 1};

/**
 * @brief Encode column data
 *
 * @param[in] chunks EncChunk device array [rowgroup][column]
 * @param[in] num_columns Number of columns
 * @param[in] num_rowgroups Number of row groups
 *
 **/
// blockDim {512,1,1}
extern "C" __global__ void __launch_bounds__(512)
  gpuEncodeOrcColumnData(EncChunk *chunks, uint32_t num_columns, uint32_t num_rowgroups)
{
  __shared__ __align__(16) orcenc_state_s state_g;

  orcenc_state_s *const s = &state_g;
  uint32_t col_id         = blockIdx.x;
  uint32_t group_id       = blockIdx.y;
  int t                   = threadIdx.x;

  if (t < sizeof(EncChunk) / sizeof(uint32_t)) {
    ((volatile uint32_t *)&s->chunk)[t] =
      ((const uint32_t *)&chunks[group_id * num_columns + col_id])[t];
  }
  if (t < CI_NUM_STREAMS) { s->strm_pos[t] = 0; }
  __syncthreads();
  if (!t) {
    s->cur_row      = 0;
    s->present_rows = 0;
    s->present_out  = 0;
    s->numvals      = 0;
    s->numlengths   = 0;
    s->nnz          = 0;
    // Dictionary data is encoded in a separate kernel
    if (s->chunk.encoding_kind == DICTIONARY_V2) {
      s->strm_pos[CI_DATA2]      = s->chunk.strm_len[CI_DATA2];
      s->strm_pos[CI_DICTIONARY] = s->chunk.strm_len[CI_DICTIONARY];
    }
  }
  __syncthreads();
  while (s->cur_row < s->chunk.num_rows || s->numvals + s->numlengths != 0) {
    // Encode valid map
    if (s->present_rows < s->chunk.num_rows) {
      uint32_t present_rows = s->present_rows;
      uint32_t nrows        = min(s->chunk.num_rows - present_rows,
                           512 * 8 - (present_rows - (min(s->cur_row, s->present_out) & ~7)));
      uint32_t nrows_out;
      if (t * 8 < nrows) {
        uint32_t row  = s->chunk.start_row + present_rows + t * 8;
        uint8_t valid = 0;
        if (row < s->chunk.valid_rows) {
          const uint8_t *valid_map_base =
            reinterpret_cast<const uint8_t *>(s->chunk.valid_map_base);
          valid = (valid_map_base) ? valid_map_base[row >> 3] : 0xff;
          if (row + 7 > s->chunk.valid_rows) {
            valid = valid & ((1 << (s->chunk.valid_rows & 7)) - 1);
          }
        }
        s->valid_buf[(row >> 3) & 0x1ff] = valid;
      }
      __syncthreads();
      present_rows += nrows;
      if (!t) { s->present_rows = present_rows; }
      // RLE encode the present stream
      nrows_out =
        present_rows -
        s->present_out;  // Should always be a multiple of 8 except at the end of the last row group
      if (nrows_out > ((present_rows < s->chunk.num_rows) ? 130 * 8 : 0)) {
        uint32_t present_out = s->present_out;
        if (s->chunk.strm_id[CI_PRESENT] >= 0) {
          uint32_t flush = (present_rows < s->chunk.num_rows) ? 0 : 7;
          nrows_out      = (nrows_out + flush) >> 3;
          nrows_out =
            ByteRLE<CI_PRESENT, 0x1ff>(
              s, s->valid_buf, (s->chunk.start_row + present_out) >> 3, nrows_out, flush, t) *
            8;
        }
        __syncthreads();
        if (!t) { s->present_out = min(present_out + nrows_out, present_rows); }
      }
      __syncthreads();
    }
    // Fetch non-null values
    if (!s->chunk.streams[CI_DATA]) {
      // Pass-through
      __syncthreads();
      if (!t) {
        s->cur_row           = s->present_rows;
        s->strm_pos[CI_DATA] = s->cur_row * s->chunk.dtype_len;
      }
      __syncthreads();
    } else if (s->cur_row < s->present_rows) {
      uint32_t maxnumvals = (s->chunk.type_kind == BOOLEAN) ? 2048 : 1024;
      uint32_t nrows =
        min(min(s->present_rows - s->cur_row, maxnumvals - max(s->numvals, s->numlengths)), 512);
      uint32_t row   = s->chunk.start_row + s->cur_row + t;
      uint32_t valid = (t < nrows) ? (s->valid_buf[(row >> 3) & 0x1ff] >> (row & 7)) & 1 : 0;
      s->buf.u32[t]  = valid;

      // TODO: Could use a faster reduction relying on _popc() for the initial phase
      lengths_to_positions(s->buf.u32, 512, t);
      __syncthreads();
      if (valid) {
        int nz_idx          = (s->nnz + s->buf.u32[t] - 1) & (maxnumvals - 1);
        const uint8_t *base = reinterpret_cast<const uint8_t *>(s->chunk.column_data_base);
        switch (s->chunk.type_kind) {
          case INT:
          case DATE:
          case FLOAT: s->vals.u32[nz_idx] = reinterpret_cast<const uint32_t *>(base)[row]; break;
          case DOUBLE:
          case LONG: s->vals.u64[nz_idx] = reinterpret_cast<const uint64_t *>(base)[row]; break;
          case SHORT: s->vals.u32[nz_idx] = reinterpret_cast<const uint16_t *>(base)[row]; break;
          case BOOLEAN:
          case BYTE: s->vals.u8[nz_idx] = reinterpret_cast<const uint8_t *>(base)[row]; break;
          case TIMESTAMP: {
            int64_t ts       = reinterpret_cast<const int64_t *>(base)[row];
            int32_t ts_scale = kTimeScale[min(s->chunk.scale, 9)];
            int64_t seconds  = ts / ts_scale;
            int32_t nanos    = (ts - seconds * ts_scale);
            // There is a bug in the ORC spec such that for negative timestamps, it is understood
            // between the writer and reader that nanos will be adjusted to their positive component
            // but the negative seconds will be left alone. This means that -2.6 is encoded as
            // seconds = -2 and nanos = 1+(-0.6) = 0.4
            // This leads to an error in decoding time where -1 < time (s) < 0
            // Details: https://github.com/rapidsai/cudf/pull/5529#issuecomment-648768925
            if (nanos < 0) { nanos += ts_scale; }
            s->vals.i64[nz_idx] = seconds - kORCTimeToUTC;
            if (nanos != 0) {
              // Trailing zeroes are encoded in the lower 3-bits
              uint32_t zeroes = 0;
              nanos *= kTimeScale[9 - min(s->chunk.scale, 9)];
              if (!(nanos % 100)) {
                nanos /= 100;
                zeroes = 1;
                while (zeroes < 7 && !(nanos % 10)) {
                  nanos /= 10;
                  zeroes++;
                }
              }
              nanos = (nanos << 3) + zeroes;
            }
            s->lengths.u32[nz_idx] = nanos;
            break;
          }
          case STRING:
            if (s->chunk.encoding_kind == DICTIONARY_V2) {
              uint32_t dict_idx = reinterpret_cast<const uint32_t *>(base)[row];
              if (dict_idx > 0x7fffffffu)
                dict_idx = reinterpret_cast<const uint32_t *>(base)[dict_idx & 0x7fffffffu];
              s->vals.u32[nz_idx] = dict_idx;
            } else {
              const nvstrdesc_s *str_desc = reinterpret_cast<const nvstrdesc_s *>(base) + row;
              const char *ptr             = str_desc->ptr;
              uint32_t count              = static_cast<uint32_t>(str_desc->count);
              s->u.strenc.str_data[s->buf.u32[t] - 1] = ptr;
              s->lengths.u32[nz_idx]                  = count;
            }
            break;
          default: break;
        }
      }
      __syncthreads();
      if (s->chunk.type_kind == STRING && s->chunk.encoding_kind != DICTIONARY_V2) {
        // Store string data
        uint32_t nz     = s->buf.u32[511];
        uint32_t nz_idx = (s->nnz + t) & 0x3ff;
        uint32_t len    = (t < nz && s->u.strenc.str_data[t]) ? s->lengths.u32[nz_idx] : 0;
        StoreStringData(s->chunk.streams[CI_DATA] + s->strm_pos[CI_DATA], &s->u.strenc, len, t);
        if (!t) { s->strm_pos[CI_DATA] += s->u.strenc.char_count; }
        __syncthreads();
      } else if (s->chunk.type_kind == BOOLEAN) {
        // bool8 -> 8x bool1
        uint32_t nz = s->buf.u32[511];
        uint8_t n   = ((s->nnz + nz) - (s->nnz & ~7) + 7) >> 3;
        if (t < n) {
          uint32_t idx8                              = (s->nnz & ~7) + (t << 3);
          s->lengths.u8[((s->nnz >> 3) + t) & 0x1ff] = ((s->vals.u8[(idx8 + 0) & 0x7ff] & 1) << 7) |
                                                       ((s->vals.u8[(idx8 + 1) & 0x7ff] & 1) << 6) |
                                                       ((s->vals.u8[(idx8 + 2) & 0x7ff] & 1) << 5) |
                                                       ((s->vals.u8[(idx8 + 3) & 0x7ff] & 1) << 4) |
                                                       ((s->vals.u8[(idx8 + 4) & 0x7ff] & 1) << 3) |
                                                       ((s->vals.u8[(idx8 + 5) & 0x7ff] & 1) << 2) |
                                                       ((s->vals.u8[(idx8 + 6) & 0x7ff] & 1) << 1) |
                                                       ((s->vals.u8[(idx8 + 7) & 0x7ff] & 1) << 0);
        }
        __syncthreads();
      }
      if (!t) {
        uint32_t nz = s->buf.u32[511];
        s->nnz += nz;
        s->numvals += nz;
        s->numlengths += (s->chunk.type_kind == TIMESTAMP ||
                          (s->chunk.type_kind == STRING && s->chunk.encoding_kind != DICTIONARY_V2))
                           ? nz
                           : 0;
        s->cur_row += nrows;
      }
      __syncthreads();
      // Encode values
      if (s->numvals > 0) {
        uint32_t flush = (s->cur_row == s->chunk.num_rows) ? 7 : 0, n;
        switch (s->chunk.type_kind) {
          case SHORT:
          case INT:
          case DATE:
            n = IntegerRLE<CI_DATA, int32_t, true, 0x3ff>(
              s, s->vals.i32, s->nnz - s->numvals, s->numvals, flush, t);
            break;
          case LONG:
          case TIMESTAMP:
            n = IntegerRLE<CI_DATA, int64_t, true, 0x3ff>(
              s, s->vals.i64, s->nnz - s->numvals, s->numvals, flush, t);
            break;
          case BYTE:
            n = ByteRLE<CI_DATA, 0x3ff>(s, s->vals.u8, s->nnz - s->numvals, s->numvals, flush, t);
            break;
          case BOOLEAN:
            n = ByteRLE<CI_DATA, 0x1ff>(s,
                                        s->lengths.u8,
                                        (s->nnz - s->numvals + flush) >> 3,
                                        (s->numvals + flush) >> 3,
                                        flush,
                                        t) *
                8;
            break;
          case FLOAT:
            StoreBytes<CI_DATA, 0xfff>(s, s->vals.u8, (s->nnz - s->numvals) * 4, s->numvals * 4, t);
            n = s->numvals;
            break;
          case DOUBLE:
            StoreBytes<CI_DATA, 0x1fff>(
              s, s->vals.u8, (s->nnz - s->numvals) * 8, s->numvals * 8, t);
            n = s->numvals;
            break;
          case STRING:
            if (s->chunk.encoding_kind == DICTIONARY_V2) {
              n = IntegerRLE<CI_DATA, uint32_t, false, 0x3ff>(
                s, s->vals.u32, s->nnz - s->numvals, s->numvals, flush, t);
            } else {
              n = s->numvals;
            }
            break;
          default: n = s->numvals; break;
        }
        __syncthreads();
        if (!t) { s->numvals -= min(n, s->numvals); }
      }
      // Encode secondary stream values
      if (s->numlengths > 0) {
        uint32_t flush = (s->cur_row == s->chunk.num_rows) ? 1 : 0, n;
        switch (s->chunk.type_kind) {
          case TIMESTAMP:
          case STRING:
            n = IntegerRLE<CI_DATA2, uint32_t, false, 0x3ff>(
              s, s->lengths.u32, s->nnz - s->numlengths, s->numlengths, flush, t);
            break;
          default: n = s->numlengths; break;
        }
        __syncthreads();
        if (!t) { s->numlengths -= min(n, s->numlengths); }
      }
    }
    __syncthreads();
  }
  __syncthreads();
  if (t <= CI_PRESENT && s->chunk.strm_id[t] >= 0) {
    // Update actual compressed length
    chunks[group_id * num_columns + col_id].strm_len[t] = s->strm_pos[t];
    if (!s->chunk.streams[t]) {
      chunks[group_id * num_columns + col_id].streams[t] =
        reinterpret_cast<uint8_t *>(const_cast<void *>(s->chunk.column_data_base)) +
        s->chunk.start_row * s->chunk.dtype_len;
    }
  }
}

/**
 * @brief Encode column dictionaries
 *
 * @param[in] stripes Stripe dictionaries device array [stripe][string_column]
 * @param[in] chunks EncChunk device array [rowgroup][column]
 * @param[in] num_columns Number of columns
 *
 **/
// blockDim {512,1,1}
extern "C" __global__ void __launch_bounds__(512)
  gpuEncodeStringDictionaries(StripeDictionary *stripes, EncChunk *chunks, uint32_t num_columns)
{
  __shared__ __align__(16) orcenc_state_s state_g;

  orcenc_state_s *const s = &state_g;
  uint32_t stripe_id      = blockIdx.x;
  uint32_t cid            = (blockIdx.y) ? CI_DICTIONARY : CI_DATA2;
  uint32_t chunk_id;
  int t = threadIdx.x;
  const nvstrdesc_s *str_desc;
  const uint32_t *dict_data;

  if (t < sizeof(StripeDictionary) / sizeof(uint32_t)) {
    ((volatile uint32_t *)&s->u.dict_stripe)[t] = ((const uint32_t *)&stripes[stripe_id])[t];
  }
  __syncthreads();
  chunk_id = s->u.dict_stripe.start_chunk * num_columns + s->u.dict_stripe.column_id;
  if (t < sizeof(EncChunk) / sizeof(uint32_t)) {
    ((volatile uint32_t *)&s->chunk)[t] = ((const uint32_t *)&chunks[chunk_id])[t];
  }
  if (t == 0) {
    s->strm_pos[cid] = 0;
    s->numlengths    = 0;
    s->nrows         = s->u.dict_stripe.num_strings;
    s->cur_row       = 0;
  }
  str_desc  = reinterpret_cast<const nvstrdesc_s *>(s->u.dict_stripe.column_data_base);
  dict_data = s->u.dict_stripe.dict_data;
  __syncthreads();
  if (s->chunk.encoding_kind != DICTIONARY_V2) {
    return;  // This column isn't using dictionary encoding -> bail out
  }

  while (s->cur_row < s->nrows || s->numlengths != 0) {
    uint32_t numvals    = min(s->nrows - s->cur_row, min(1024 - s->numlengths, 512));
    uint32_t string_idx = (t < numvals) ? dict_data[s->cur_row + t] : 0;
    if (cid == CI_DICTIONARY) {
      // Encoding string contents
      const char *ptr = (t < numvals) ? str_desc[string_idx].ptr : 0;
      uint32_t count  = (t < numvals) ? static_cast<uint32_t>(str_desc[string_idx].count) : 0;
      s->u.strenc.str_data[t] = ptr;
      StoreStringData(s->chunk.streams[CI_DICTIONARY] + s->strm_pos[CI_DICTIONARY],
                      &s->u.strenc,
                      (ptr) ? count : 0,
                      t);
      if (!t) { s->strm_pos[CI_DICTIONARY] += s->u.strenc.char_count; }
    } else {
      // Encoding string lengths
      uint32_t count  = (t < numvals) ? static_cast<uint32_t>(str_desc[string_idx].count) : 0;
      uint32_t nz_idx = (s->cur_row + t) & 0x3ff;
      if (t < numvals) s->lengths.u32[nz_idx] = count;
      __syncthreads();
      if (s->numlengths + numvals > 0) {
        uint32_t flush = (s->cur_row + numvals == s->nrows) ? 1 : 0;
        uint32_t n     = IntegerRLE<CI_DATA2, uint32_t, false, 0x3ff>(
          s, s->lengths.u32, s->cur_row, s->numlengths + numvals, flush, t);
        __syncthreads();
        if (!t) {
          s->numlengths += numvals;
          s->numlengths -= min(n, s->numlengths);
        }
      }
    }
    if (t == 0) { s->cur_row += numvals; }
    __syncthreads();
  }
  if (t == 0) { chunks[chunk_id].strm_len[cid] = s->strm_pos[cid]; }
}

/**
 * @brief Merge chunked column data into a single contiguous stream
 *
 * @param[in] strm_desc StripeStream device array [stripe][stream]
 * @param[in] chunks EncChunk device array [rowgroup][column]
 * @param[in] num_stripe_streams Total number of streams
 * @param[in] num_columns Number of columns
 *
 **/
// blockDim {1024,1,1}
extern "C" __global__ void __launch_bounds__(1024)
  gpuCompactOrcDataStreams(StripeStream *strm_desc, EncChunk *chunks, uint32_t num_columns)
{
  __shared__ __align__(16) StripeStream ss;
  __shared__ __align__(16) EncChunk ck0;
  __shared__ uint8_t *volatile ck_curptr_g;
  __shared__ uint32_t volatile ck_curlen_g;

  uint32_t strm_id = blockIdx.x;
  uint32_t ck0_id, cid;
  uint32_t t = threadIdx.x;
  uint8_t *dst_ptr;

  if (t < sizeof(StripeStream) / sizeof(uint32_t)) {
    ((volatile uint32_t *)&ss)[t] = ((const uint32_t *)&strm_desc[strm_id])[t];
  }
  __syncthreads();
  ck0_id = ss.first_chunk_id;
  if (t < sizeof(EncChunk) / sizeof(uint32_t)) {
    ((volatile uint32_t *)&ck0)[t] = ((const uint32_t *)&chunks[ck0_id])[t];
  }
  __syncthreads();
  cid     = ss.stream_type;
  dst_ptr = ck0.streams[cid] + ck0.strm_len[cid];
  for (uint32_t g = 1; g < ss.num_chunks; g++) {
    uint8_t *src_ptr;
    uint32_t len;
    if (t == 0) {
      src_ptr = chunks[ck0_id + g * num_columns].streams[cid];
      len     = chunks[ck0_id + g * num_columns].strm_len[cid];
      if (src_ptr != dst_ptr) { chunks[ck0_id + g * num_columns].streams[cid] = dst_ptr; }
      ck_curptr_g = src_ptr;
      ck_curlen_g = len;
    }
    __syncthreads();
    src_ptr = ck_curptr_g;
    len     = ck_curlen_g;
    if (len > 0 && src_ptr != dst_ptr) {
      for (uint32_t i = 0; i < len; i += 1024) {
        uint8_t v = (i + t < len) ? src_ptr[i + t] : 0;
        __syncthreads();
        if (i + t < len) { dst_ptr[i + t] = v; }
      }
    }
    dst_ptr += len;
    __syncthreads();
  }
  if (!t) { strm_desc[strm_id].stream_size = dst_ptr - ck0.streams[cid]; }
}

/**
 * @brief Initializes compression input/output structures
 *
 * @param[in] strm_desc StripeStream device array [stripe][stream]
 * @param[in] chunks EncChunk device array [rowgroup][column]
 * @param[out] comp_in Per-block compression input parameters
 * @param[out] comp_out Per-block compression status
 * @param[in] compressed_bfr Compression output buffer
 * @param[in] comp_blk_size Compression block size
 *
 **/
// blockDim {256,1,1}
extern "C" __global__ void __launch_bounds__(256)
  gpuInitCompressionBlocks(StripeStream *strm_desc,
                           EncChunk *chunks,
                           gpu_inflate_input_s *comp_in,
                           gpu_inflate_status_s *comp_out,
                           uint8_t *compressed_bfr,
                           uint32_t comp_blk_size)
{
  __shared__ __align__(16) StripeStream ss;
  __shared__ uint8_t *volatile uncomp_base_g;

  uint32_t strm_id = blockIdx.x;
  uint32_t t       = threadIdx.x;
  uint32_t num_blocks;
  uint8_t *src, *dst;

  if (t < sizeof(StripeStream) / sizeof(uint32_t)) {
    ((volatile uint32_t *)&ss)[t] = ((const uint32_t *)&strm_desc[strm_id])[t];
  }
  __syncthreads();
  if (t == 0) { uncomp_base_g = chunks[ss.first_chunk_id].streams[ss.stream_type]; }
  __syncthreads();
  src        = uncomp_base_g;
  dst        = compressed_bfr + ss.bfr_offset;
  num_blocks = (ss.stream_size > 0) ? (ss.stream_size - 1) / comp_blk_size + 1 : 1;
  for (uint32_t b = t; b < num_blocks; b += 256) {
    gpu_inflate_input_s *blk_in   = &comp_in[ss.first_block + b];
    gpu_inflate_status_s *blk_out = &comp_out[ss.first_block + b];
    uint32_t blk_size = min(comp_blk_size, ss.stream_size - min(b * comp_blk_size, ss.stream_size));
    blk_in->srcDevice = src + b * comp_blk_size;
    blk_in->srcSize   = blk_size;
    blk_in->dstDevice = dst + b * (3 + comp_blk_size) + 3;  // reserve 3 bytes for block header
    blk_in->dstSize   = blk_size;
    blk_out->bytes_written = blk_size;
    blk_out->status        = 1;
    blk_out->reserved      = 0;
  }
}

/**
 * @brief Compacts compressed blocks in a single contiguous stream, and update 3-byte block length
 *fields
 *
 * @param[in,out] strm_desc StripeStream device array [stripe][stream]
 * @param[in] chunks EncChunk device array [rowgroup][column]
 * @param[in] comp_in Per-block compression input parameters
 * @param[in] comp_out Per-block compression status
 * @param[in] compressed_bfr Compression output buffer
 * @param[in] comp_blk_size Compression block size
 *
 **/
// blockDim {1024,1,1}
extern "C" __global__ void __launch_bounds__(1024)
  gpuCompactCompressedBlocks(StripeStream *strm_desc,
                             gpu_inflate_input_s *comp_in,
                             gpu_inflate_status_s *comp_out,
                             uint8_t *compressed_bfr,
                             uint32_t comp_blk_size)
{
  __shared__ __align__(16) StripeStream ss;
  __shared__ const uint8_t *volatile comp_src_g;
  __shared__ uint32_t volatile comp_len_g;

  uint32_t strm_id = blockIdx.x;
  uint32_t t       = threadIdx.x;
  uint32_t num_blocks, b, blk_size;
  const uint8_t *src;
  uint8_t *dst;

  if (t < sizeof(StripeStream) / sizeof(uint32_t)) {
    ((volatile uint32_t *)&ss)[t] = ((const uint32_t *)&strm_desc[strm_id])[t];
  }
  __syncthreads();
  num_blocks = (ss.stream_size > 0) ? (ss.stream_size - 1) / comp_blk_size + 1 : 0;
  dst        = compressed_bfr + ss.bfr_offset;
  b          = 0;
  do {
    if (t == 0) {
      gpu_inflate_input_s *blk_in   = &comp_in[ss.first_block + b];
      gpu_inflate_status_s *blk_out = &comp_out[ss.first_block + b];
      uint32_t src_len =
        min(comp_blk_size, ss.stream_size - min(b * comp_blk_size, ss.stream_size));
      uint32_t dst_len = (blk_out->status == 0) ? blk_out->bytes_written : src_len;
      uint32_t blk_size24;
      if (dst_len >= src_len) {
        // Copy from uncompressed source
        src                    = reinterpret_cast<const uint8_t *>(blk_in->srcDevice);
        blk_out->bytes_written = src_len;
        dst_len                = src_len;
        blk_size24             = dst_len * 2 + 1;
      } else {
        // Compressed block
        src        = reinterpret_cast<const uint8_t *>(blk_in->dstDevice);
        blk_size24 = dst_len * 2 + 0;
      }
      dst[0]     = static_cast<uint8_t>(blk_size24 >> 0);
      dst[1]     = static_cast<uint8_t>(blk_size24 >> 8);
      dst[2]     = static_cast<uint8_t>(blk_size24 >> 16);
      comp_src_g = src;
      comp_len_g = dst_len;
    }
    __syncthreads();
    src      = comp_src_g;
    blk_size = comp_len_g;
    dst += 3;  // skip over length written by thread0
    if (src != dst) {
      for (uint32_t i = 0; i < blk_size; i += 1024) {
        uint8_t v = (i + t < blk_size) ? src[i + t] : 0;
        __syncthreads();
        if (i + t < blk_size) { dst[i + t] = v; }
      }
    }
    dst += blk_size;
    __syncthreads();
  } while (++b < num_blocks);
  // Update stripe stream with the compressed size
  if (t == 0) {
    strm_desc[strm_id].stream_size = static_cast<uint32_t>(dst - (compressed_bfr + ss.bfr_offset));
  }
}

/**
 * @brief Launches kernel for encoding column data
 *
 * @param[in] chunks EncChunk device array [rowgroup][column]
 * @param[in] num_columns Number of columns
 * @param[in] num_rowgroups Number of row groups
 * @param[in] stream CUDA stream to use, default 0
 *
 * @return hipSuccess if successful, a CUDA error code otherwise
 **/
hipError_t EncodeOrcColumnData(EncChunk *chunks,
                                uint32_t num_columns,
                                uint32_t num_rowgroups,
                                hipStream_t stream)
{
  dim3 dim_block(512, 1);  // 512 threads per chunk
  dim3 dim_grid(num_columns, num_rowgroups);
  gpuEncodeOrcColumnData<<<dim_grid, dim_block, 0, stream>>>(chunks, num_columns, num_rowgroups);
  return hipSuccess;
}

/**
 * @brief Launches kernel for encoding column dictionaries
 *
 * @param[in] stripes Stripe dictionaries device array [stripe][string_column]
 * @param[in] chunks EncChunk device array [rowgroup][column]
 * @param[in] num_string_columns Number of string columns
 * @param[in] num_columns Number of columns
 * @param[in] num_stripes Number of stripes
 * @param[in] stream CUDA stream to use, default 0
 *
 * @return hipSuccess if successful, a CUDA error code otherwise
 **/
hipError_t EncodeStripeDictionaries(StripeDictionary *stripes,
                                     EncChunk *chunks,
                                     uint32_t num_string_columns,
                                     uint32_t num_columns,
                                     uint32_t num_stripes,
                                     hipStream_t stream)
{
  dim3 dim_block(512, 1);  // 512 threads per dictionary
  dim3 dim_grid(num_string_columns * num_stripes, 2);
  gpuEncodeStringDictionaries<<<dim_grid, dim_block, 0, stream>>>(stripes, chunks, num_columns);
  return hipSuccess;
}

/**
 * @brief Launches kernel for compacting chunked column data prior to compression
 *
 * @param[in] strm_desc StripeStream device array [stripe][stream]
 * @param[in] chunks EncChunk device array [rowgroup][column]
 * @param[in] num_stripe_streams Total number of streams
 * @param[in] num_columns Number of columns
 * @param[in] stream CUDA stream to use, default 0
 *
 * @return hipSuccess if successful, a CUDA error code otherwise
 **/
hipError_t CompactOrcDataStreams(StripeStream *strm_desc,
                                  EncChunk *chunks,
                                  uint32_t num_stripe_streams,
                                  uint32_t num_columns,
                                  hipStream_t stream)
{
  dim3 dim_block(1024, 1);
  dim3 dim_grid(num_stripe_streams, 1);
  gpuCompactOrcDataStreams<<<dim_grid, dim_block, 0, stream>>>(strm_desc, chunks, num_columns);
  return hipSuccess;
}

/**
 * @brief Launches kernel(s) for compressing data streams
 *
 * @param[in] compressed_data Output compressed blocks
 * @param[in] strm_desc StripeStream device array [stripe][stream]
 * @param[in] chunks EncChunk device array [rowgroup][column]
 * @param[out] comp_in Per-block compression input parameters
 * @param[out] comp_out Per-block compression status
 * @param[in] num_stripe_streams Total number of streams
 * @param[in] num_compressed_blocks Total number of compressed blocks
 * @param[in] compression Type of compression
 * @param[in] comp_blk_size Compression block size
 * @param[in] stream CUDA stream to use, default 0
 *
 * @return hipSuccess if successful, a CUDA error code otherwise
 **/
hipError_t CompressOrcDataStreams(uint8_t *compressed_data,
                                   StripeStream *strm_desc,
                                   EncChunk *chunks,
                                   gpu_inflate_input_s *comp_in,
                                   gpu_inflate_status_s *comp_out,
                                   uint32_t num_stripe_streams,
                                   uint32_t num_compressed_blocks,
                                   CompressionKind compression,
                                   uint32_t comp_blk_size,
                                   hipStream_t stream)
{
  dim3 dim_block_init(256, 1);
  dim3 dim_grid(num_stripe_streams, 1);
  gpuInitCompressionBlocks<<<dim_grid, dim_block_init, 0, stream>>>(
    strm_desc, chunks, comp_in, comp_out, compressed_data, comp_blk_size);
  if (compression == SNAPPY) { gpu_snap(comp_in, comp_out, num_compressed_blocks, stream); }
  dim3 dim_block_compact(1024, 1);
  gpuCompactCompressedBlocks<<<dim_grid, dim_block_compact, 0, stream>>>(
    strm_desc, comp_in, comp_out, compressed_data, comp_blk_size);
  return hipSuccess;
}

}  // namespace gpu
}  // namespace orc
}  // namespace io
}  // namespace cudf
