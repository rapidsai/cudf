#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019-2020, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "orc_common.h"
#include "orc_gpu.h"

#include <io/utilities/block_utils.cuh>

#include <rmm/cuda_stream_view.hpp>

namespace cudf {
namespace io {
namespace orc {
namespace gpu {
struct compressed_stream_s {
  CompressedStreamInfo info;
  gpu_inflate_input_s ctl;
};

// blockDim {128,1,1}
extern "C" __global__ void __launch_bounds__(128, 8) gpuParseCompressedStripeData(
  CompressedStreamInfo *strm_info, int32_t num_streams, uint32_t block_size, uint32_t log2maxcr)
{
  __shared__ compressed_stream_s strm_g[4];

  compressed_stream_s *const s = &strm_g[threadIdx.x / 32];
  int strm_id                  = blockIdx.x * 4 + (threadIdx.x / 32);
  int lane_id                  = threadIdx.x % 32;

  if (lane_id == 0) { s->info = strm_info[strm_id]; }

  __syncthreads();
  if (strm_id < num_streams) {
    // Walk through the compressed blocks
    const uint8_t *cur               = s->info.compressed_data;
    const uint8_t *end               = cur + s->info.compressed_data_size;
    uint8_t *uncompressed            = s->info.uncompressed_data;
    size_t max_uncompressed_size     = 0;
    uint32_t num_compressed_blocks   = 0;
    uint32_t num_uncompressed_blocks = 0;
    while (cur + 3 < end) {
      uint32_t block_len = shuffle((lane_id == 0) ? cur[0] | (cur[1] << 8) | (cur[2] << 16) : 0);
      uint32_t is_uncompressed = block_len & 1;
      uint32_t uncompressed_size;
      gpu_inflate_input_s *init_ctl = nullptr;
      block_len >>= 1;
      cur += 3;
      if (block_len > block_size || cur + block_len > end) {
        // Fatal
        num_compressed_blocks = 0;
        max_uncompressed_size = 0;
        break;
      }
      // TBD: For some codecs like snappy, it wouldn't be too difficult to get the actual
      // uncompressed size and avoid waste due to block size alignment For now, rely on the max
      // compression ratio to limit waste for the most extreme cases (small single-block streams)
      uncompressed_size =
        (is_uncompressed)
          ? block_len
          : (block_len < (block_size >> log2maxcr)) ? block_len << log2maxcr : block_size;
      if (is_uncompressed) {
        if (uncompressed_size <= 32) {
          // For short blocks, copy the uncompressed data to output
          if (uncompressed &&
              max_uncompressed_size + uncompressed_size <= s->info.max_uncompressed_size &&
              lane_id < uncompressed_size) {
            uncompressed[max_uncompressed_size + lane_id] = cur[lane_id];
          }
        } else {
          init_ctl = s->info.copyctl;
          init_ctl = (init_ctl && num_uncompressed_blocks < s->info.num_uncompressed_blocks)
                       ? &init_ctl[num_uncompressed_blocks]
                       : nullptr;
          num_uncompressed_blocks++;
        }
      } else {
        init_ctl = s->info.decctl;
        init_ctl = (init_ctl && num_compressed_blocks < s->info.num_compressed_blocks)
                     ? &init_ctl[num_compressed_blocks]
                     : nullptr;
        num_compressed_blocks++;
      }
      if (!lane_id && init_ctl) {
        s->ctl.srcDevice = const_cast<uint8_t *>(cur);
        s->ctl.srcSize   = block_len;
        s->ctl.dstDevice = uncompressed + max_uncompressed_size;
        s->ctl.dstSize   = uncompressed_size;
      }
      __syncwarp();
      if (init_ctl && lane_id == 0) *init_ctl = s->ctl;
      cur += block_len;
      max_uncompressed_size += uncompressed_size;
    }
    __syncwarp();
    if (!lane_id) {
      s->info.num_compressed_blocks   = num_compressed_blocks;
      s->info.num_uncompressed_blocks = num_uncompressed_blocks;
      s->info.max_uncompressed_size   = max_uncompressed_size;
    }
  }

  __syncthreads();
  if (strm_id < num_streams && lane_id == 0) strm_info[strm_id] = s->info;
}

// blockDim {128,1,1}
extern "C" __global__ void __launch_bounds__(128, 8)
  gpuPostDecompressionReassemble(CompressedStreamInfo *strm_info, int32_t num_streams)
{
  __shared__ compressed_stream_s strm_g[4];

  compressed_stream_s *const s = &strm_g[threadIdx.x / 32];
  int strm_id                  = blockIdx.x * 4 + (threadIdx.x / 32);
  int lane_id                  = threadIdx.x % 32;

  if (strm_id < num_streams && lane_id == 0) s->info = strm_info[strm_id];

  __syncthreads();
  if (strm_id < num_streams &&
      s->info.num_compressed_blocks + s->info.num_uncompressed_blocks > 0 &&
      s->info.max_uncompressed_size > 0) {
    // Walk through the compressed blocks
    const uint8_t *cur                  = s->info.compressed_data;
    const uint8_t *end                  = cur + s->info.compressed_data_size;
    const gpu_inflate_input_s *dec_in   = s->info.decctl;
    const gpu_inflate_status_s *dec_out = s->info.decstatus;
    uint8_t *uncompressed_actual        = s->info.uncompressed_data;
    uint8_t *uncompressed_estimated     = uncompressed_actual;
    uint32_t num_compressed_blocks      = 0;
    uint32_t max_compressed_blocks      = s->info.num_compressed_blocks;

    while (cur + 3 < end) {
      uint32_t block_len = shuffle((lane_id == 0) ? cur[0] | (cur[1] << 8) | (cur[2] << 16) : 0);
      uint32_t is_uncompressed = block_len & 1;
      uint32_t uncompressed_size_est, uncompressed_size_actual;
      block_len >>= 1;
      cur += 3;
      if (cur + block_len > end) { break; }
      if (is_uncompressed) {
        uncompressed_size_est    = block_len;
        uncompressed_size_actual = block_len;
      } else {
        if (num_compressed_blocks > max_compressed_blocks) { break; }
        if (shuffle((lane_id == 0) ? dec_out[num_compressed_blocks].status : 0) != 0) {
          // Decompression failed, not much point in doing anything else
          break;
        }
        uncompressed_size_est =
          shuffle((lane_id == 0) ? *(const uint32_t *)&dec_in[num_compressed_blocks].dstSize : 0);
        uncompressed_size_actual = shuffle(
          (lane_id == 0) ? *(const uint32_t *)&dec_out[num_compressed_blocks].bytes_written : 0);
      }
      // In practice, this should never happen with a well-behaved writer, as we would expect the
      // uncompressed size to always be equal to the compression block size except for the last
      // block
      if (uncompressed_actual < uncompressed_estimated) {
        // warp-level memmove
        for (int i = lane_id; i < (int)uncompressed_size_actual; i += 32) {
          uncompressed_actual[i] = uncompressed_estimated[i];
        }
      }
      cur += block_len;
      num_compressed_blocks += 1 - is_uncompressed;
      uncompressed_estimated += uncompressed_size_est;
      uncompressed_actual += uncompressed_size_actual;
    }
    // Update info with actual uncompressed size
    if (!lane_id) {
      size_t total_uncompressed_size = uncompressed_actual - s->info.uncompressed_data;
      // Set uncompressed size to zero if there were any errors
      strm_info[strm_id].max_uncompressed_size =
        (num_compressed_blocks == s->info.num_compressed_blocks) ? total_uncompressed_size : 0;
    }
  }
}

/**
 * @brief Shared mem state for gpuParseRowGroupIndex
 *
 */
struct rowindex_state_s {
  ColumnDesc chunk;
  uint32_t rowgroup_start;
  uint32_t rowgroup_end;
  int is_compressed;
  uint32_t row_index_entry[3][CI_PRESENT];  // NOTE: Assumes CI_PRESENT follows CI_DATA and CI_DATA2
  CompressedStreamInfo strm_info[2];
  RowGroup rowgroups[128];
  uint32_t compressed_offset[128][2];
};

enum row_entry_state_e {
  NOT_FOUND = 0,
  GET_LENGTH,
  SKIP_VARINT,
  SKIP_FIXEDLEN,
  STORE_INDEX0,
  STORE_INDEX1,
  STORE_INDEX2,
};

/**
 * @brief Decode a single row group index entry
 *
 * @param[in,out] s row group index state
 * @param[in] start start position in byte stream
 * @param[in] end end of byte stream
 * @return bytes consumed
 *
 **/
static uint32_t __device__ ProtobufParseRowIndexEntry(rowindex_state_s *s,
                                                      const uint8_t *start,
                                                      const uint8_t *end)
{
  constexpr uint32_t pb_rowindexentry_id = static_cast<uint32_t>(PB_TYPE_FIXEDLEN) + 8;

  const uint8_t *cur      = start;
  row_entry_state_e state = NOT_FOUND;
  uint32_t length = 0, strm_idx_id = s->chunk.skip_count >> 8, idx_id = 1, ci_id = CI_PRESENT,
           pos_end = 0;
  while (cur < end) {
    uint32_t v = 0;
    for (uint32_t l = 0; l <= 28; l += 7) {
      uint32_t c = (cur < end) ? *cur++ : 0;
      v |= (c & 0x7f) << l;
      if (c <= 0x7f) break;
    }
    switch (state) {
      case NOT_FOUND:
        if (v == pb_rowindexentry_id) {
          state = GET_LENGTH;
        } else {
          v &= 7;
          if (v == PB_TYPE_FIXED64)
            cur += 8;
          else if (v == PB_TYPE_FIXED32)
            cur += 4;
          else if (v == PB_TYPE_VARINT)
            state = SKIP_VARINT;
          else if (v == PB_TYPE_FIXEDLEN)
            state = SKIP_FIXEDLEN;
        }
        break;
      case SKIP_VARINT: state = NOT_FOUND; break;
      case SKIP_FIXEDLEN:
        cur += v;
        state = NOT_FOUND;
        break;
      case GET_LENGTH:
        if (length == 0) {
          length = (uint32_t)(cur + v - start);
          state = NOT_FOUND;  // Scan for positions (same field id & low-level type as RowIndexEntry
                              // entry)
        } else {
          pos_end = min((uint32_t)(cur + v - start), length);
          state   = STORE_INDEX0;
        }
        break;
      case STORE_INDEX0:
        ci_id = (idx_id == (strm_idx_id & 0xff))
                  ? CI_DATA
                  : (idx_id == ((strm_idx_id >> 8) & 0xff)) ? CI_DATA2 : CI_PRESENT;
        idx_id++;
        if (s->is_compressed) {
          if (ci_id < CI_PRESENT) s->row_index_entry[0][ci_id] = v;
          if (cur >= start + pos_end) return length;
          state = STORE_INDEX1;
          break;
        } else {
          if (ci_id < CI_PRESENT) s->row_index_entry[0][ci_id] = 0;
          // Fall through to STORE_INDEX1 for uncompressed (always block0)
        }
      case STORE_INDEX1:
        if (ci_id < CI_PRESENT) s->row_index_entry[1][ci_id] = v;
        if (cur >= start + pos_end) return length;
        state = (ci_id == CI_DATA && s->chunk.encoding_kind != DICTIONARY &&
                 s->chunk.encoding_kind != DICTIONARY_V2 &&
                 (s->chunk.type_kind == STRING || s->chunk.type_kind == BINARY ||
                  s->chunk.type_kind == VARCHAR || s->chunk.type_kind == CHAR ||
                  s->chunk.type_kind == DECIMAL || s->chunk.type_kind == FLOAT ||
                  s->chunk.type_kind == DOUBLE))
                  ? STORE_INDEX0
                  : STORE_INDEX2;
        break;
      case STORE_INDEX2:
        if (ci_id < CI_PRESENT) {
          // Boolean columns have an extra byte to indicate the position of the bit within the byte
          s->row_index_entry[2][ci_id] = (s->chunk.type_kind == BOOLEAN) ? (v << 3) + *cur : v;
        }
        if (ci_id == CI_PRESENT || s->chunk.type_kind == BOOLEAN) cur++;
        if (cur >= start + pos_end) return length;
        state = STORE_INDEX0;
        break;
    }
  }
  return (uint32_t)(end - start);
}

/**
 * @brief Decode row group index entries
 *
 * @param[in,out] s row group index state
 * @param[in] num_rowgroups Number of index entries to read
 *
 **/
static __device__ void gpuReadRowGroupIndexEntries(rowindex_state_s *s, int num_rowgroups)
{
  const uint8_t *index_data = s->chunk.streams[CI_INDEX];
  int index_data_len        = s->chunk.strm_len[CI_INDEX];
  for (int i = 0; i < num_rowgroups; i++) {
    s->row_index_entry[0][0] = 0;
    s->row_index_entry[0][1] = 0;
    s->row_index_entry[1][0] = 0;
    s->row_index_entry[1][1] = 0;
    s->row_index_entry[2][0] = 0;
    s->row_index_entry[2][1] = 0;
    if (index_data_len > 0) {
      int len = ProtobufParseRowIndexEntry(s, index_data, index_data + index_data_len);
      index_data += len;
      index_data_len = max(index_data_len - len, 0);
      for (int j = 0; j < 2; j++) {
        s->rowgroups[i].strm_offset[j] = s->row_index_entry[1][j];
        s->rowgroups[i].run_pos[j]     = s->row_index_entry[2][j];
        s->compressed_offset[i][j]     = s->row_index_entry[0][j];
      }
    }
  }
  s->chunk.streams[CI_INDEX]  = index_data;
  s->chunk.strm_len[CI_INDEX] = index_data_len;
}

/**
 * @brief Translate block+offset compressed position into an uncompressed offset
 *
 * @param[in,out] s row group index state
 * @param[in] ci_id index to convert (CI_DATA or CI_DATA2)
 * @param[in] num_rowgroups Number of index entries
 * @param[in] t thread id
 *
 **/
static __device__ void gpuMapRowIndexToUncompressed(rowindex_state_s *s,
                                                    int ci_id,
                                                    int num_rowgroups,
                                                    int t)
{
  int32_t strm_len = s->chunk.strm_len[ci_id];
  if (strm_len > 0) {
    int32_t compressed_offset = (t < num_rowgroups) ? s->compressed_offset[t][ci_id] : 0;
    if (compressed_offset > 0) {
      const uint8_t *start            = s->strm_info[ci_id].compressed_data;
      const uint8_t *cur              = start;
      const uint8_t *end              = cur + s->strm_info[ci_id].compressed_data_size;
      gpu_inflate_status_s *decstatus = s->strm_info[ci_id].decstatus;
      uint32_t uncomp_offset          = 0;
      for (;;) {
        uint32_t block_len, is_uncompressed;

        if (cur + 3 > end || cur + 3 >= start + compressed_offset) { break; }
        block_len = cur[0] | (cur[1] << 8) | (cur[2] << 16);
        cur += 3;
        is_uncompressed = block_len & 1;
        block_len >>= 1;
        cur += block_len;
        if (cur > end) { break; }
        if (is_uncompressed) {
          uncomp_offset += block_len;
        } else {
          uncomp_offset += decstatus->bytes_written;
          decstatus++;
        }
      }
      s->rowgroups[t].strm_offset[ci_id] += uncomp_offset;
    }
  }
}

/**
 * @brief Decode index streams
 *
 * @param[out] row_groups RowGroup device array [rowgroup][column]
 * @param[in] strm_info List of compressed streams (or NULL if uncompressed)
 * @param[in] chunks ColumnDesc device array [stripe][column]
 * @param[in] num_columns Number of columns
 * @param[in] num_stripes Number of stripes
 * @param[in] num_rowgroups Number of row groups
 *
 **/
// blockDim {128,1,1}
extern "C" __global__ void __launch_bounds__(128, 8)
  gpuParseRowGroupIndex(RowGroup *row_groups,
                        CompressedStreamInfo *strm_info,
                        ColumnDesc *chunks,
                        uint32_t num_columns,
                        uint32_t num_stripes,
                        uint32_t num_rowgroups,
                        uint32_t rowidx_stride)
{
  __shared__ __align__(16) rowindex_state_s state_g;
  rowindex_state_s *const s = &state_g;
  uint32_t chunk_id         = blockIdx.y * num_columns + blockIdx.x;
  int t                     = threadIdx.x;

  if (t == 0) {
    s->chunk = chunks[chunk_id];
    if (strm_info) {
      if (s->chunk.strm_len[0] > 0) s->strm_info[0] = strm_info[s->chunk.strm_id[0]];
      if (s->chunk.strm_len[1] > 0) s->strm_info[1] = strm_info[s->chunk.strm_id[1]];
    }

    uint32_t rowgroups_in_chunk =
      (rowidx_stride > 0) ? (s->chunk.num_rows + rowidx_stride - 1) / rowidx_stride : 1;
    s->rowgroup_start = s->chunk.rowgroup_id;
    s->rowgroup_end   = s->rowgroup_start + rowgroups_in_chunk;
    s->is_compressed  = (strm_info != NULL);
  }
  __syncthreads();
  while (s->rowgroup_start < s->rowgroup_end) {
    int num_rowgroups = min(s->rowgroup_end - s->rowgroup_start, 128);
    int rowgroup_size4, t4, t32;

    s->rowgroups[t].chunk_id = chunk_id;
    if (t == 0) { gpuReadRowGroupIndexEntries(s, num_rowgroups); }
    __syncthreads();
    if (s->is_compressed) {
      // Convert the block + blk_offset pair into a raw offset into the decompressed stream
      if (s->chunk.strm_len[CI_DATA] > 0) {
        gpuMapRowIndexToUncompressed(s, CI_DATA, num_rowgroups, t);
      }
      if (s->chunk.strm_len[CI_DATA2] > 0) {
        gpuMapRowIndexToUncompressed(s, CI_DATA2, num_rowgroups, t);
      }
      __syncthreads();
    }
    rowgroup_size4 = sizeof(RowGroup) / sizeof(uint32_t);
    t4             = t & 3;
    t32            = t >> 2;
    for (int i = t32; i < num_rowgroups; i += 32) {
      for (int j = t4; j < rowgroup_size4; j += 4) {
        ((uint32_t *)&row_groups[(s->rowgroup_start + i) * num_columns + blockIdx.x])[j] =
          ((volatile uint32_t *)&s->rowgroups[i])[j];
      }
    }
    __syncthreads();
    if (t == 0) { s->rowgroup_start += num_rowgroups; }
    __syncthreads();
  }
}

void __host__ ParseCompressedStripeData(CompressedStreamInfo *strm_info,
                                        int32_t num_streams,
                                        uint32_t compression_block_size,
                                        uint32_t log2maxcr,
                                        rmm::cuda_stream_view stream)
{
  dim3 dim_block(128, 1);
  dim3 dim_grid((num_streams + 3) >> 2, 1);  // 1 stream per warp, 4 warps per block
  gpuParseCompressedStripeData<<<dim_grid, dim_block, 0, stream.value()>>>(
    strm_info, num_streams, compression_block_size, log2maxcr);
}

void __host__ PostDecompressionReassemble(CompressedStreamInfo *strm_info,
                                          int32_t num_streams,
                                          rmm::cuda_stream_view stream)
{
  dim3 dim_block(128, 1);
  dim3 dim_grid((num_streams + 3) >> 2, 1);  // 1 stream per warp, 4 warps per block
  gpuPostDecompressionReassemble<<<dim_grid, dim_block, 0, stream.value()>>>(strm_info,
                                                                             num_streams);
}

/**
 * @brief Launches kernel for constructing rowgroup from index streams
 *
 * @param[out] row_groups RowGroup device array [rowgroup][column]
 * @param[in] strm_info List of compressed streams (or NULL if uncompressed)
 * @param[in] chunks ColumnDesc device array [stripe][column]
 * @param[in] num_columns Number of columns
 * @param[in] num_stripes Number of stripes
 * @param[in] num_rowgroups Number of row groups
 * @param[in] stream CUDA stream to use, default 0
 */
void __host__ ParseRowGroupIndex(RowGroup *row_groups,
                                 CompressedStreamInfo *strm_info,
                                 ColumnDesc *chunks,
                                 uint32_t num_columns,
                                 uint32_t num_stripes,
                                 uint32_t num_rowgroups,
                                 uint32_t rowidx_stride,
                                 rmm::cuda_stream_view stream)
{
  dim3 dim_block(128, 1);
  dim3 dim_grid(num_columns, num_stripes);  // 1 column chunk per block
  gpuParseRowGroupIndex<<<dim_grid, dim_block, 0, stream.value()>>>(
    row_groups, strm_info, chunks, num_columns, num_stripes, num_rowgroups, rowidx_stride);
}

}  // namespace gpu
}  // namespace orc
}  // namespace io
}  // namespace cudf
