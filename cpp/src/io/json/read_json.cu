#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2022-2025, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "io/comp/io_uncomp.hpp"
#include "io/json/nested_json.hpp"
#include "io/utilities/getenv_or.hpp"
#include "read_json.hpp"

#include <cudf/concatenate.hpp>
#include <cudf/detail/nvtx/ranges.hpp>
#include <cudf/detail/utilities/cuda_memcpy.hpp>
#include <cudf/detail/utilities/integer_utils.hpp>
#include <cudf/detail/utilities/stream_pool.hpp>
#include <cudf/detail/utilities/vector_factories.hpp>
#include <cudf/io/datasource.hpp>
#include <cudf/io/detail/json.hpp>
#include <cudf/utilities/error.hpp>
#include <cudf/utilities/memory_resource.hpp>
#include <cudf/utilities/span.hpp>

#include <rmm/cuda_stream_pool.hpp>
#include <rmm/device_uvector.hpp>
#include <rmm/exec_policy.hpp>

#include <thrust/distance.h>
#include <thrust/execution_policy.h>
#include <thrust/iterator/constant_iterator.h>
#include <thrust/scatter.h>

#include <BS_thread_pool.hpp>
#include <BS_thread_pool_utils.hpp>

#include <functional>
#include <numeric>

namespace cudf::io::json::detail {

namespace {

namespace pools {

BS::thread_pool& tpool()
{
  static std::size_t pool_size =
    getenv_or("LIBCUDF_HOST_COMPRESSION_NUM_THREADS", std::thread::hardware_concurrency());
  static BS::thread_pool _tpool(pool_size);
  return _tpool;
}

}  // namespace pools

class compressed_host_buffer_source final : public datasource {
 public:
  explicit compressed_host_buffer_source(std::unique_ptr<datasource> const& src,
                                         compression_type comptype)
    : _comptype{comptype}, _dbuf_ptr{src->host_read(0, src->size())}
  {
    auto ch_buffer = host_span<uint8_t const>(reinterpret_cast<uint8_t const*>(_dbuf_ptr->data()),
                                              _dbuf_ptr->size());
    if (_comptype == compression_type::GZIP || _comptype == compression_type::ZIP ||
        _comptype == compression_type::SNAPPY) {
      _decompressed_ch_buffer_size = cudf::io::detail::get_uncompressed_size(_comptype, ch_buffer);
    } else {
      _decompressed_buffer         = cudf::io::detail::decompress(_comptype, ch_buffer);
      _decompressed_ch_buffer_size = _decompressed_buffer.size();
    }
  }

  std::size_t host_read(std::size_t offset, std::size_t size, uint8_t* dst) override
  {
    auto ch_buffer = host_span<uint8_t const>(reinterpret_cast<uint8_t const*>(_dbuf_ptr->data()),
                                              _dbuf_ptr->size());
    if (_decompressed_buffer.empty()) {
      auto decompressed_hbuf = cudf::io::detail::decompress(_comptype, ch_buffer);
      auto const count       = std::min(size, decompressed_hbuf.size() - offset);
      bool partial_read      = offset + count < decompressed_hbuf.size();
      if (!partial_read) {
        std::memcpy(dst, decompressed_hbuf.data() + offset, count);
        return count;
      }
      _decompressed_buffer = std::move(decompressed_hbuf);
    }
    auto const count = std::min(size, _decompressed_buffer.size() - offset);
    std::memcpy(dst, _decompressed_buffer.data() + offset, count);
    return count;
  }

  std::unique_ptr<buffer> host_read(std::size_t offset, std::size_t size) override
  {
    auto ch_buffer = host_span<uint8_t const>(reinterpret_cast<uint8_t const*>(_dbuf_ptr->data()),
                                              _dbuf_ptr->size());
    if (_decompressed_buffer.empty()) {
      auto decompressed_hbuf = cudf::io::detail::decompress(_comptype, ch_buffer);
      auto const count       = std::min(size, decompressed_hbuf.size() - offset);
      bool partial_read      = offset + count < decompressed_hbuf.size();
      if (!partial_read)
        return std::make_unique<owning_buffer<std::vector<uint8_t>>>(
          std::move(decompressed_hbuf), decompressed_hbuf.data() + offset, count);
      _decompressed_buffer = std::move(decompressed_hbuf);
    }
    auto const count = std::min(size, _decompressed_buffer.size() - offset);
    return std::make_unique<non_owning_buffer>(_decompressed_buffer.data() + offset, count);
  }

  std::future<std::size_t> device_read_async(std::size_t offset,
                                             std::size_t size,
                                             uint8_t* dst,
                                             rmm::cuda_stream_view stream) override
  {
    auto& thread_pool = pools::tpool();
    return thread_pool.submit_task([this, offset, size, dst, stream] {
      auto hbuf = host_read(offset, size);
      CUDF_CUDA_TRY(
        hipMemcpyAsync(dst, hbuf->data(), hbuf->size(), hipMemcpyHostToDevice, stream.value()));
      stream.synchronize();
      return hbuf->size();
    });
  }

  [[nodiscard]] bool supports_device_read() const override { return true; }

  [[nodiscard]] std::size_t size() const override { return _decompressed_ch_buffer_size; }

 private:
  std::unique_ptr<datasource::buffer> _dbuf_ptr;
  compression_type _comptype;
  std::size_t _decompressed_ch_buffer_size;
  std::vector<std::uint8_t> _decompressed_buffer;
};

// Return total size of sources enclosing the passed range
std::size_t sources_size(host_span<std::unique_ptr<datasource>> const sources,
                         std::size_t range_offset,
                         std::size_t range_size)
{
  return std::accumulate(sources.begin(), sources.end(), 0ul, [=](std::size_t sum, auto& source) {
    auto const size = source->size();
    // TODO take care of 0, 0, or *, 0 case.
    return sum +
           (range_size == 0 or range_offset + range_size > size ? size - range_offset : range_size);
  });
}

// Return estimated size of subchunk using a heuristic involving the byte range size and the minimum
// subchunk size
std::size_t estimate_size_per_subchunk(std::size_t chunk_size)
{
  auto geometric_mean = [](double a, double b) { return std::sqrt(a * b); };
  // NOTE: heuristic for choosing subchunk size: geometric mean of minimum subchunk size (set to
  // 10kb) and the byte range size
  return geometric_mean(std::ceil(static_cast<double>(chunk_size) / num_subchunks),
                        min_subchunk_size);
}

/**
 * @brief Return the batch size for the JSON reader.
 *
 * The datasources passed to the JSON reader are read iteratively in batches demarcated by byte
 * range offsets. The tokenizer requires the JSON buffer read in each batch to be of size at most
 * INT_MAX bytes.
 * Since the byte range corresponding to a given batch can cause the last JSON line
 * in the batch to be incomplete, the batch size returned by this function allows for an additional
 * `max_subchunks_prealloced` subchunks to be allocated beyond the byte range offsets. Since the
 * size of the subchunk depends on the size of the byte range, the batch size is variable and cannot
 * be directly controlled by the user. As a workaround, the environment variable
 * LIBCUDF_JSON_BATCH_SIZE can be used to set a fixed batch size at runtime.
 *
 * @return size in bytes
 */
std::size_t get_batch_size(std::size_t chunk_size)
{
  auto const size_per_subchunk = estimate_size_per_subchunk(chunk_size);
  auto const batch_limit       = static_cast<std::size_t>(std::numeric_limits<int32_t>::max()) -
                           (max_subchunks_prealloced * size_per_subchunk);
  return std::min(batch_limit, getenv_or<std::size_t>("LIBCUDF_JSON_BATCH_SIZE", batch_limit));
}

/**
 * @brief Extract the first delimiter character position in the string
 *
 * @param d_data Device span in which to search for delimiter character
 * @param delimiter Delimiter character to search for
 * @param stream CUDA stream used for device memory operations and kernel launches
 *
 * @return Position of first delimiter character in device array
 */
size_type find_first_delimiter(device_span<char const> d_data,
                               char const delimiter,
                               rmm::cuda_stream_view stream)
{
  auto const first_delimiter_position =
    thrust::find(rmm::exec_policy(stream), d_data.begin(), d_data.end(), delimiter);
  return first_delimiter_position != d_data.end()
           ? static_cast<size_type>(thrust::distance(d_data.begin(), first_delimiter_position))
           : -1;
}

/**
 * @brief Get the byte range between record starts and ends starting from the given range. The
 * actual byte range read and returned will contain complete JSONL records, and will include the
 * delimiter at the end of the last record.
 *
 * if get_byte_range_offset == 0, then we can skip the first delimiter search
 * if get_byte_range_offset != 0, then we need to search for the first delimiter in given range.
 * if not found, skip this chunk, if found, then search for first delimiter in next range until we
 * find a delimiter. Use this as actual range for parsing. If the size of actual byte range to be
 * parsed is greater than the integer limit (or the requested batch size), then split the ingested
 * buffer in two. Note that as long as no single record in the JSONL input is of size larger than
 * the requested batch size, we are guaranteed that each of the two buffers will be within the batch
 * size limit - the size of the first buffer is capped at the batch limit by the batching logic
 * itself, and the second buffer contains only the last record which was incomplete in the initial
 * byte range requested. If the size of the actual byte range to be parsed does not exceed batch
 * limits, then the second buffer is empty.
 *
 * @param sources Data sources to read from
 * @param reader_opts JSON reader options with range offset and range size
 * @param stream CUDA stream used for device memory operations and kernel launches
 * @returns A pair of data source owning buffers together enclosing the bytes read. The second
 * buffer may or may not be empty depending on the condition described above.
 */
std::pair<datasource::owning_buffer<rmm::device_buffer>,
          std::optional<datasource::owning_buffer<rmm::device_buffer>>>
get_record_range_raw_input(host_span<std::unique_ptr<datasource>> sources,
                           json_reader_options const& reader_opts,
                           rmm::cuda_stream_view stream)
{
  CUDF_FUNC_RANGE();

  std::size_t const total_source_size = sources_size(sources, 0, 0);
  auto constexpr num_delimiter_chars  = 1;
  auto const delimiter                = reader_opts.get_delimiter();
  auto const num_extra_delimiters     = num_delimiter_chars * sources.size();
  std::size_t const chunk_offset      = reader_opts.get_byte_range_offset();
  std::size_t const chunk_size        = reader_opts.get_byte_range_size();
  // Sanity checks for the byte range offset and size are handled by the batching logic.
  // We only need to check if we are reading until the end of the last source in this function.
  auto const should_load_till_last_source = chunk_offset + chunk_size == total_source_size;

  int num_subchunks_prealloced        = should_load_till_last_source ? 0 : max_subchunks_prealloced;
  std::size_t const size_per_subchunk = estimate_size_per_subchunk(chunk_size);

  std::size_t buffer_size =
    std::min(total_source_size, chunk_size + num_subchunks_prealloced * size_per_subchunk) +
    num_extra_delimiters;
  rmm::device_buffer buffer(buffer_size, stream);
  device_span<char> bufspan(reinterpret_cast<char*>(buffer.data()), buffer.size());

  // Offset within buffer indicating first read position
  std::int64_t buffer_offset = 0;
  auto readbufspan =
    ingest_raw_input(bufspan, sources, chunk_offset, chunk_size, delimiter, stream);
  auto const requested_size = readbufspan.size();

  auto const shift_for_nonzero_offset = std::min<std::int64_t>(chunk_offset, 1);
  auto const first_delim_pos =
    chunk_offset == 0 ? 0 : find_first_delimiter(readbufspan, delimiter, stream);

  // If we read till the end of the last source, we cannot be sure
  // if the last record read ends with a delimiter. In such cases, we add a delimiter
  // nevertheless; even if the record terminates
  // with a delimiter, adding a extra delimiter does not affect the table constructed since the
  // parser ignores empty lines.
  auto insert_delimiter = [delimiter, stream](device_span<char> subspan) {
    auto last_char = delimiter;
    cudf::detail::cuda_memcpy<char>(subspan, host_span<char const>(&last_char, 1, false), stream);
  };

  // If the requested byte range ends with a delimiter at the end of line n, we will still need to
  // continue reading since the next batch begins at the start of the n+1^th record and skips the
  // entire line until the first delimiter is encountered at the end of the line.
  if (first_delim_pos == -1) {
    // return empty owning datasource buffer
    auto empty_buf = rmm::device_buffer(0, stream);
    return std::make_pair(datasource::owning_buffer<rmm::device_buffer>(std::move(empty_buf)),
                          std::nullopt);
  } else if (!should_load_till_last_source) {
    // Find next delimiter
    std::int64_t next_delim_pos     = -1;
    std::size_t next_subchunk_start = chunk_offset + chunk_size;
    while (next_delim_pos < buffer_offset) {
      for (int subchunk = 0;
           subchunk < num_subchunks_prealloced && next_delim_pos < buffer_offset &&
           next_subchunk_start < total_source_size;
           subchunk++) {
        buffer_offset += readbufspan.size();
        readbufspan    = ingest_raw_input(bufspan.last(buffer_size - buffer_offset),
                                       sources,
                                       next_subchunk_start,
                                       size_per_subchunk,
                                       delimiter,
                                       stream);
        next_delim_pos = find_first_delimiter(readbufspan, delimiter, stream) + buffer_offset;
        next_subchunk_start += size_per_subchunk;
      }
      if (next_delim_pos < buffer_offset) {
        if (next_subchunk_start >= total_source_size) {
          // If we have reached the end of source list but the source does not terminate with a
          // delimiter character
          next_delim_pos = buffer_offset + readbufspan.size();
          insert_delimiter(bufspan.subspan(next_delim_pos, 1));
        } else {
          // Reallocate-and-retry policy
          // Our buffer_size estimate is insufficient to read until the end of the line! We need to
          // allocate more memory and try again!
          num_subchunks_prealloced *= 2;
          buffer_size = std::min(total_source_size,
                                 buffer_size + num_subchunks_prealloced * size_per_subchunk) +
                        num_extra_delimiters;
          buffer.resize(buffer_size, stream);
          bufspan = device_span<char>(reinterpret_cast<char*>(buffer.data()), buffer.size());
        }
      }
    }

    // If the size of the ingested buffer is less than the batch size, we can simply return the
    // buffer as is, and set the optional second buffer to null.
    // If the size of the ingested buffer exceed the batch size limits due to the
    // reallocate-and-retry policy, we split the ingested buffer in two parts. The second part
    // only contains the last record in the buffer, while the first part contains all the remaining
    // lines.
    // As long as the size of no record exceeds the batch size limit placed, we are guaranteed that
    // the returned buffer(s) will be below the batch limit.
    auto const batch_size = getenv_or<std::size_t>(
      "LIBCUDF_JSON_BATCH_SIZE", static_cast<std::size_t>(std::numeric_limits<int32_t>::max()));
    if (static_cast<std::size_t>(next_delim_pos - first_delim_pos - shift_for_nonzero_offset) <
        batch_size) {
      return std::make_pair(
        datasource::owning_buffer<rmm::device_buffer>(
          std::move(buffer),
          reinterpret_cast<uint8_t*>(buffer.data()) + first_delim_pos + shift_for_nonzero_offset,
          next_delim_pos - first_delim_pos - shift_for_nonzero_offset + 1),
        std::nullopt);
    }
    device_span<char const> bufsubspan =
      bufspan.subspan(first_delim_pos + shift_for_nonzero_offset,
                      requested_size - first_delim_pos - shift_for_nonzero_offset);
    auto rev_it_begin = thrust::make_reverse_iterator(bufsubspan.end());
    auto rev_it_end   = thrust::make_reverse_iterator(bufsubspan.begin());
    auto const second_last_delimiter_it =
      thrust::find(rmm::exec_policy(stream), rev_it_begin, rev_it_end, delimiter);
    CUDF_EXPECTS(second_last_delimiter_it != rev_it_end,
                 "A single JSON line cannot be larger than the batch size limit");
    auto const last_line_size =
      next_delim_pos - requested_size +
      static_cast<std::size_t>(thrust::distance(rev_it_begin, second_last_delimiter_it));
    CUDF_EXPECTS(last_line_size < batch_size,
                 "A single JSON line cannot be larger than the batch size limit");

    rmm::device_buffer second_buffer(bufsubspan.data() + static_cast<std::size_t>(thrust::distance(
                                                           second_last_delimiter_it, rev_it_end)),
                                     last_line_size + 1,
                                     stream);

    return std::make_pair(
      datasource::owning_buffer<rmm::device_buffer>(
        std::move(buffer),
        reinterpret_cast<uint8_t*>(buffer.data()) + first_delim_pos + shift_for_nonzero_offset,
        next_delim_pos - first_delim_pos - shift_for_nonzero_offset - last_line_size),
      datasource::owning_buffer<rmm::device_buffer>(
        std::move(second_buffer),
        reinterpret_cast<uint8_t*>(second_buffer.data()),
        second_buffer.size()));
  }

  // Add delimiter to end of buffer - possibly adding an empty line to the input buffer - iff we are
  // reading till the end of the last source i.e. should_load_till_last_source is true. Note that
  // the table generated from the JSONL input remains unchanged since empty lines are ignored by the
  // parser.
  std::size_t num_chars = readbufspan.size() - first_delim_pos - shift_for_nonzero_offset;
  if (num_chars) {
    insert_delimiter(bufspan.subspan(readbufspan.size(), 1));
    num_chars++;
  }

  return std::make_pair(
    datasource::owning_buffer<rmm::device_buffer>(
      std::move(buffer),
      reinterpret_cast<uint8_t*>(buffer.data()) + first_delim_pos + shift_for_nonzero_offset,
      num_chars),
    std::nullopt);
}

/**
 * @brief Helper function to read the current batch using the byte range offsets and size
 * passed, normalize it, and construct a partial table.
 */
std::pair<table_with_metadata, std::optional<table_with_metadata>> read_batch(
  host_span<std::unique_ptr<datasource>> sources,
  json_reader_options const& reader_opts,
  rmm::cuda_stream_view stream,
  rmm::device_async_resource_ref mr)
{
  CUDF_FUNC_RANGE();
  // The second owning buffer in the pair returned by get_record_range_raw_input may not be
  // populated depending on the size of the actual byte range read. The first owning buffer will
  // always be non-empty.
  auto owning_buffers = get_record_range_raw_input(sources, reader_opts, stream);

  // If input JSON buffer has single quotes and option to normalize single quotes is enabled,
  // invoke pre-processing FST
  if (reader_opts.is_enabled_normalize_single_quotes()) {
    normalize_single_quotes(owning_buffers.first,
                            reader_opts.get_delimiter(),
                            stream,
                            cudf::get_current_device_resource_ref());
    stream.synchronize();
  }

  auto buffer = cudf::device_span<char const>(
    reinterpret_cast<char const*>(owning_buffers.first.data()), owning_buffers.first.size());
  auto first_partial_table = device_parse_nested_json(buffer, reader_opts, stream, mr);
  if (!owning_buffers.second.has_value())
    return std::make_pair(std::move(first_partial_table), std::nullopt);

  // Repeat the normalization and table construction steps for the second buffer if it exists
  if (reader_opts.is_enabled_normalize_single_quotes()) {
    normalize_single_quotes(owning_buffers.second.value(),
                            reader_opts.get_delimiter(),
                            stream,
                            cudf::get_current_device_resource_ref());
    stream.synchronize();
  }
  buffer = cudf::device_span<char const>(
    reinterpret_cast<char const*>(owning_buffers.second.value().data()),
    owning_buffers.second.value().size());
  auto second_partial_table = device_parse_nested_json(buffer, reader_opts, stream, mr);
  return std::make_pair(std::move(first_partial_table), std::move(second_partial_table));
}

/**
 * @brief Helper function that implements the batching logic for the JSONL reader.
 * The goal of the batched reader is to handle reading multiple JSONL sources whose total cumulative
 * size exceeds the integer limit imposed by the JSON tokenizer. The batching logic divides the
 * requested input byte range spanning sources into smaller batches, each of which itself spans
 * multiple sources. The batches are constructed such that the byte subrange in each batch does not
 * exceed the batch size, which is either set using the environment variable
 * LIBCUDF_JSON_BATCH_SIZE, or is set to a little under the integer limit. Note that batching
 * sources does not work for for regular JSON inputs.
 */
table_with_metadata read_json_impl(host_span<std::unique_ptr<datasource>> sources,
                                   json_reader_options const& reader_opts,
                                   rmm::cuda_stream_view stream,
                                   rmm::device_async_resource_ref mr)
{
  std::size_t const total_source_size = sources_size(sources, 0, 0);

  // Batching is enabled only for JSONL inputs, not regular JSON files
  CUDF_EXPECTS(
    reader_opts.is_enabled_lines() || total_source_size < std::numeric_limits<int32_t>::max(),
    "Parsing Regular JSON inputs of size greater than INT_MAX bytes is not supported");

  // Sanity checks of byte range offset and clamping of byte range size
  std::size_t const chunk_offset = reader_opts.get_byte_range_offset();
  CUDF_EXPECTS(total_source_size ? chunk_offset < total_source_size : !chunk_offset,
               "Invalid byte range offset",
               std::invalid_argument);
  std::size_t chunk_size       = reader_opts.get_byte_range_size();
  chunk_size                   = !chunk_size ? total_source_size - chunk_offset
                                             : std::min(chunk_size, total_source_size - chunk_offset);
  std::size_t const batch_size = get_batch_size(chunk_size);

  // Identify the position (zero-indexed) of starting source file from which to begin
  // batching based on byte range offset. If the offset is larger than the sum of all
  // source sizes, then start_source is total number of source files i.e. no file is
  // read.
  // Prefix sum of source file sizes
  std::size_t pref_source_size = 0;
  // Starting source file from which to being batching evaluated using byte range offset
  std::size_t const start_source = [chunk_offset, &sources, &pref_source_size]() {
    for (std::size_t src_idx = 0; src_idx < sources.size(); ++src_idx) {
      if (pref_source_size + sources[src_idx]->size() > chunk_offset) { return src_idx; }
      pref_source_size += sources[src_idx]->size();
    }
    return sources.size();
  }();
  // Construct batches of byte ranges spanning source files, with the starting position of batches
  // indicated by `batch_offsets`. `pref_bytes_size` gives the bytes position from which the current
  // batch begins, and `end_bytes_size` gives the terminal bytes position after which reading
  // stops.
  std::size_t pref_bytes_size = chunk_offset;
  std::size_t end_bytes_size  = chunk_offset + chunk_size;
  std::vector<std::size_t> batch_offsets{pref_bytes_size};
  for (std::size_t i = start_source; i < sources.size() && pref_bytes_size < end_bytes_size;) {
    pref_source_size += sources[i]->size();
    // If the current source file can subsume multiple batches, we split the file until the
    // boundary of the last batch exceeds the end of the file (indexed by `pref_source_size`)
    while (pref_bytes_size < end_bytes_size &&
           pref_source_size >= std::min(pref_bytes_size + batch_size, end_bytes_size)) {
      auto next_batch_size = std::min(batch_size, end_bytes_size - pref_bytes_size);
      batch_offsets.push_back(batch_offsets.back() + next_batch_size);
      pref_bytes_size += next_batch_size;
    }
    i++;
  }

  std::vector<cudf::io::table_with_metadata> partial_tables;
  json_reader_options batched_reader_opts{reader_opts};
  batched_reader_opts.set_byte_range_offset(chunk_offset);
  batched_reader_opts.set_byte_range_size(chunk_size);

  // lambda to insert the partial tables into the vector. Since read_batch function returns a pair
  // of partial tables where the second table is optional, we insert a table into the vector only if
  // it is non-empty
  auto insert_partial_tables =
    [&partial_tables](
      std::pair<table_with_metadata, std::optional<table_with_metadata>>&& partial_table_pair) {
      if (partial_table_pair.first.tbl->num_columns() == 0 &&
          partial_table_pair.first.tbl->num_rows() == 0)
        return false;
      partial_tables.emplace_back(std::move(partial_table_pair.first));
      if (partial_table_pair.second.has_value()) {
        if (partial_table_pair.second.value().tbl->num_columns() == 0 &&
            partial_table_pair.second.value().tbl->num_rows() == 0)
          return false;
        partial_tables.emplace_back(std::move(partial_table_pair.second.value()));
      }
      return true;
    };

  // recursive lambda to construct schema_element. Here, we assume that the table from the
  // first batch contains all the columns in the concatenated table, and that the partial tables
  // from all following batches contain the same set of columns
  std::function<schema_element(cudf::host_span<column_view const> cols,
                               cudf::host_span<column_name_info const> names,
                               schema_element & schema)>
    construct_schema;
  schema_element schema{data_type{cudf::type_id::STRUCT}};
  construct_schema = [&construct_schema](cudf::host_span<column_view const> children,
                                         cudf::host_span<column_name_info const> children_props,
                                         schema_element& schema) -> schema_element {
    CUDF_EXPECTS(
      children.size() == children_props.size(),
      "Mismatch in the number of children columns and children column properties received");

    if (schema.type == data_type{cudf::type_id::LIST}) {
      schema.column_order = {"element"};
      CUDF_EXPECTS(children.size() == 2, "List should have two children");
      auto element_idx = children_props[0].name == "element" ? 0 : 1;
      schema_element child_schema{children[element_idx].type()};
      std::vector<column_view> grandchildren_cols;
      std::transform(children[element_idx].child_begin(),
                     children[element_idx].child_end(),
                     std::back_inserter(grandchildren_cols),
                     [](auto& gc) { return gc; });
      schema.child_types["element"] =
        construct_schema(grandchildren_cols, children_props[element_idx].children, child_schema);
    } else {
      std::vector<std::string> col_order;
      std::transform(children_props.begin(),
                     children_props.end(),
                     std::back_inserter(col_order),
                     [](auto& c_prop) { return c_prop.name; });
      schema.column_order = std::move(col_order);
      for (auto i = 0ul; i < children.size(); i++) {
        schema_element child_schema{children[i].type()};
        std::vector<column_view> grandchildren_cols;
        std::transform(children[i].child_begin(),
                       children[i].child_end(),
                       std::back_inserter(grandchildren_cols),
                       [](auto& gc) { return gc; });
        schema.child_types[children_props[i].name] =
          construct_schema(grandchildren_cols, children_props[i].children, child_schema);
      }
    }

    return schema;
  };

  if (batch_offsets.size() <= 2) {
    // single batch
    auto has_inserted = insert_partial_tables(
      read_batch(sources, batched_reader_opts, stream, cudf::get_current_device_resource_ref()));
    if (!has_inserted) {
      return table_with_metadata{std::make_unique<table>(std::vector<std::unique_ptr<column>>{}),
                                 {std::vector<column_name_info>{}}};
    }
  } else {
    // multiple batches
    batched_reader_opts.set_byte_range_offset(batch_offsets[0]);
    batched_reader_opts.set_byte_range_size(batch_offsets[1] - batch_offsets[0]);
    insert_partial_tables(
      read_batch(sources, batched_reader_opts, stream, cudf::get_current_device_resource_ref()));

    auto& tbl = partial_tables.back().tbl;
    std::vector<column_view> children;
    for (size_type j = 0; j < tbl->num_columns(); j++) {
      children.emplace_back(tbl->get_column(j));
    }
    batched_reader_opts.set_dtypes(
      construct_schema(children, partial_tables.back().metadata.schema_info, schema));
    batched_reader_opts.enable_prune_columns(true);

    // Dispatch individual batches to read_batch and push the resulting table into
    // partial_tables array. Note that the reader options need to be updated for each
    // batch to adjust byte range offset and byte range size.
    for (std::size_t batch_offset_pos = 1; batch_offset_pos < batch_offsets.size() - 1;
         batch_offset_pos++) {
      batched_reader_opts.set_byte_range_offset(batch_offsets[batch_offset_pos]);
      batched_reader_opts.set_byte_range_size(batch_offsets[batch_offset_pos + 1] -
                                              batch_offsets[batch_offset_pos]);
      auto has_inserted = insert_partial_tables(
        read_batch(sources, batched_reader_opts, stream, cudf::get_current_device_resource_ref()));

      if (!has_inserted) {
        CUDF_EXPECTS(batch_offset_pos == batch_offsets.size() - 2,
                     "Only the partial table generated by the last batch can be empty");
        break;
      }
    }
  }

  // If there is a single partial table, then there is no need to concatenate
  if (partial_tables.size() == 1) return std::move(partial_tables[0]);
  auto expects_schema_equality =
    std::all_of(partial_tables.begin() + 1,
                partial_tables.end(),
                [&gt = partial_tables[0].metadata.schema_info](auto& ptbl) {
                  return ptbl.metadata.schema_info == gt;
                });
  CUDF_EXPECTS(expects_schema_equality,
               "Mismatch in JSON schema across batches in multi-source multi-batch reading");

  auto partial_table_views = std::vector<cudf::table_view>(partial_tables.size());
  std::transform(partial_tables.begin(),
                 partial_tables.end(),
                 partial_table_views.begin(),
                 [](auto const& table) { return table.tbl->view(); });
  return table_with_metadata{cudf::concatenate(partial_table_views, stream, mr),
                             {partial_tables[0].metadata.schema_info}};
}

}  // anonymous namespace

device_span<char> ingest_raw_input(device_span<char> buffer,
                                   host_span<std::unique_ptr<datasource>> sources,
                                   std::size_t range_offset,
                                   std::size_t range_size,
                                   char delimiter,
                                   rmm::cuda_stream_view stream)
{
  CUDF_FUNC_RANGE();
  // We append a line delimiter between two files to make sure the last line of file i and the first
  // line of file i+1 don't end up on the same JSON line, if file i does not already end with a line
  // delimiter.
  auto constexpr num_delimiter_chars = 1;
  std::vector<std::future<std::size_t>> thread_tasks;

  auto delimiter_map = cudf::detail::make_empty_host_vector<std::size_t>(sources.size(), stream);
  std::vector<std::size_t> prefsum_source_sizes(sources.size());
  std::vector<std::unique_ptr<datasource::buffer>> h_buffers;
  std::size_t bytes_read = 0;
  std::transform_inclusive_scan(sources.begin(),
                                sources.end(),
                                prefsum_source_sizes.begin(),
                                std::plus<std::size_t>{},
                                [](std::unique_ptr<datasource> const& s) { return s->size(); });
  auto upper =
    std::upper_bound(prefsum_source_sizes.begin(), prefsum_source_sizes.end(), range_offset);
  std::size_t start_source = std::distance(prefsum_source_sizes.begin(), upper);

  auto const total_bytes_to_read = std::min(range_size, prefsum_source_sizes.back() - range_offset);
  range_offset -= start_source ? prefsum_source_sizes[start_source - 1] : 0;

  std::size_t const num_streams =
    std::min<std::size_t>({sources.size() - start_source + 1,
                           cudf::detail::global_cuda_stream_pool().get_stream_pool_size(),
                           pools::tpool().get_thread_count()});
  auto stream_pool = cudf::detail::fork_streams(stream, num_streams);
  for (std::size_t i = start_source, cur_stream = 0;
       i < sources.size() && bytes_read < total_bytes_to_read;
       i++) {
    if (sources[i]->is_empty()) continue;
    auto data_size = std::min(sources[i]->size() - range_offset, total_bytes_to_read - bytes_read);
    auto destination = reinterpret_cast<uint8_t*>(buffer.data()) + bytes_read +
                       (num_delimiter_chars * delimiter_map.size());
    if (sources[i]->supports_device_read()) {
      thread_tasks.emplace_back(sources[i]->device_read_async(
        range_offset, data_size, destination, stream_pool[cur_stream++ % stream_pool.size()]));
      bytes_read += data_size;
    } else {
      h_buffers.emplace_back(sources[i]->host_read(range_offset, data_size));
      auto const& h_buffer = h_buffers.back();
      CUDF_CUDA_TRY(hipMemcpyAsync(
        destination, h_buffer->data(), h_buffer->size(), hipMemcpyHostToDevice, stream.value()));
      bytes_read += h_buffer->size();
    }
    range_offset = 0;
    delimiter_map.push_back(bytes_read + (num_delimiter_chars * delimiter_map.size()));
  }
  // Removing delimiter inserted after last non-empty source is read
  if (!delimiter_map.empty()) { delimiter_map.pop_back(); }

  // If this is a multi-file source, we scatter the JSON line delimiters between files
  if (sources.size() > 1 && !delimiter_map.empty()) {
    static_assert(num_delimiter_chars == 1,
                  "Currently only single-character delimiters are supported");
    auto const delimiter_source = thrust::make_constant_iterator(delimiter);
    auto const d_delimiter_map  = cudf::detail::make_device_uvector_async(
      delimiter_map, stream, cudf::get_current_device_resource_ref());
    thrust::scatter(rmm::exec_policy_nosync(stream),
                    delimiter_source,
                    delimiter_source + d_delimiter_map.size(),
                    d_delimiter_map.data(),
                    buffer.data());
  }
  stream.synchronize();

  if (thread_tasks.size()) {
    auto const bytes_read = std::accumulate(
      thread_tasks.begin(), thread_tasks.end(), std::size_t{0}, [](std::size_t sum, auto& task) {
        return sum + task.get();
      });
    CUDF_EXPECTS(bytes_read == total_bytes_to_read,
                 "Incorrect number of bytes read by multithreaded reader");
  }

  return buffer.first(bytes_read + (delimiter_map.size() * num_delimiter_chars));
}

table_with_metadata read_json(host_span<std::unique_ptr<datasource>> sources,
                              json_reader_options const& reader_opts,
                              rmm::cuda_stream_view stream,
                              rmm::device_async_resource_ref mr)
{
  CUDF_FUNC_RANGE();

  if (reader_opts.get_byte_range_offset() != 0 or reader_opts.get_byte_range_size() != 0) {
    CUDF_EXPECTS(reader_opts.is_enabled_lines(),
                 "Specifying a byte range is supported only for JSON Lines");
  }

  if (sources.size() > 1) {
    CUDF_EXPECTS(reader_opts.is_enabled_lines(),
                 "Multiple inputs are supported only for JSON Lines format");
  }

  if (reader_opts.get_compression() == compression_type::NONE)
    return read_json_impl(sources, reader_opts, stream, mr);

  std::vector<std::unique_ptr<datasource>> compressed_sources;
  std::vector<std::future<std::unique_ptr<compressed_host_buffer_source>>> thread_tasks;
  auto& thread_pool = pools::tpool();
  for (auto& src : sources) {
    thread_tasks.emplace_back(thread_pool.submit_task([&reader_opts, &src] {
      return std::make_unique<compressed_host_buffer_source>(src, reader_opts.get_compression());
    }));
  }
  std::transform(thread_tasks.begin(),
                 thread_tasks.end(),
                 std::back_inserter(compressed_sources),
                 [](auto& task) { return task.get(); });
  // in read_json_impl, we need the compressed source size to actually be the
  // uncompressed source size for correct batching
  return read_json_impl(compressed_sources, reader_opts, stream, mr);
}

}  // namespace cudf::io::json::detail
