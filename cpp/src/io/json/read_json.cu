#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2022-2024, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "io/comp/io_uncomp.hpp"
#include "io/json/legacy/read_json.hpp"
#include "io/json/nested_json.hpp"
#include "read_json.hpp"

#include <cudf/detail/nvtx/ranges.hpp>
#include <cudf/detail/utilities/stream_pool.hpp>
#include <cudf/detail/utilities/vector_factories.hpp>
#include <cudf/io/detail/json.hpp>
#include <cudf/utilities/error.hpp>
#include <cudf/utilities/span.hpp>

#include <rmm/device_uvector.hpp>
#include <rmm/exec_policy.hpp>
#include <rmm/resource_ref.hpp>

#include <thrust/iterator/constant_iterator.h>
#include <thrust/scatter.h>

#include <numeric>

namespace cudf::io::json::detail {

size_t sources_size(host_span<std::unique_ptr<datasource>> const sources,
                    size_t range_offset,
                    size_t range_size)
{
  return std::accumulate(sources.begin(), sources.end(), 0ul, [=](size_t sum, auto& source) {
    auto const size = source->size();
    // TODO take care of 0, 0, or *, 0 case.
    return sum +
           (range_size == 0 or range_offset + range_size > size ? size - range_offset : range_size);
  });
}

/**
 * @brief Read from array of data sources into RMM buffer. The size of the returned device span
          can be larger than the number of bytes requested from the list of sources when
          the range to be read spans across multiple sources. This is due to the delimiter
          characters inserted after the end of each accessed source.
 *
 * @param buffer Device span buffer to which data is read
 * @param sources Array of data sources
 * @param compression Compression format of source
 * @param range_offset Number of bytes to skip from source start
 * @param range_size Number of bytes to read from source
 * @param stream CUDA stream used for device memory operations and kernel launches
 * @returns A subspan of the input device span containing data read
 */
device_span<char> ingest_raw_input(device_span<char> buffer,
                                   host_span<std::unique_ptr<datasource>> sources,
                                   compression_type compression,
                                   size_t range_offset,
                                   size_t range_size,
                                   rmm::cuda_stream_view stream)
{
  CUDF_FUNC_RANGE();
  // We append a line delimiter between two files to make sure the last line of file i and the first
  // line of file i+1 don't end up on the same JSON line, if file i does not already end with a line
  // delimiter.
  auto constexpr num_delimiter_chars = 1;

  if (compression == compression_type::NONE) {
    std::vector<size_type> delimiter_map{};
    std::vector<size_t> prefsum_source_sizes(sources.size());
    std::vector<std::unique_ptr<datasource::buffer>> h_buffers;
    delimiter_map.reserve(sources.size());
    size_t bytes_read = 0;
    std::transform_inclusive_scan(sources.begin(),
                                  sources.end(),
                                  prefsum_source_sizes.begin(),
                                  std::plus<int>{},
                                  [](const std::unique_ptr<datasource>& s) { return s->size(); });
    auto upper =
      std::upper_bound(prefsum_source_sizes.begin(), prefsum_source_sizes.end(), range_offset);
    size_t start_source = std::distance(prefsum_source_sizes.begin(), upper);

    auto const total_bytes_to_read =
      std::min(range_size, prefsum_source_sizes.back() - range_offset);
    range_offset -= start_source ? prefsum_source_sizes[start_source - 1] : 0;
    for (size_t i = start_source; i < sources.size() && bytes_read < total_bytes_to_read; i++) {
      if (sources[i]->is_empty()) continue;
      auto data_size =
        std::min(sources[i]->size() - range_offset, total_bytes_to_read - bytes_read);
      auto destination = reinterpret_cast<uint8_t*>(buffer.data()) + bytes_read +
                         (num_delimiter_chars * delimiter_map.size());
      if (sources[i]->is_device_read_preferred(data_size)) {
        bytes_read += sources[i]->device_read(range_offset, data_size, destination, stream);
      } else {
        h_buffers.emplace_back(sources[i]->host_read(range_offset, data_size));
        auto const& h_buffer = h_buffers.back();
        CUDF_CUDA_TRY(hipMemcpyAsync(
          destination, h_buffer->data(), h_buffer->size(), hipMemcpyHostToDevice, stream.value()));
        bytes_read += h_buffer->size();
      }
      range_offset = 0;
      delimiter_map.push_back(bytes_read + (num_delimiter_chars * delimiter_map.size()));
    }
    // Removing delimiter inserted after last non-empty source is read
    if (!delimiter_map.empty()) { delimiter_map.pop_back(); }

    // If this is a multi-file source, we scatter the JSON line delimiters between files
    if (sources.size() > 1) {
      static_assert(num_delimiter_chars == 1,
                    "Currently only single-character delimiters are supported");
      auto const delimiter_source = thrust::make_constant_iterator('\n');
      auto const d_delimiter_map  = cudf::detail::make_device_uvector_async(
        delimiter_map, stream, rmm::mr::get_current_device_resource());
      thrust::scatter(rmm::exec_policy_nosync(stream),
                      delimiter_source,
                      delimiter_source + d_delimiter_map.size(),
                      d_delimiter_map.data(),
                      buffer.data());
    }
    stream.synchronize();
    return buffer.first(bytes_read + (delimiter_map.size() * num_delimiter_chars));
  }
  // TODO: allow byte range reading from multiple compressed files.
  auto remaining_bytes_to_read = std::min(range_size, sources[0]->size() - range_offset);
  auto hbuffer                 = std::vector<uint8_t>(remaining_bytes_to_read);
  // Single read because only a single compressed source is supported
  // Reading to host because decompression of a single block is much faster on the CPU
  sources[0]->host_read(range_offset, remaining_bytes_to_read, hbuffer.data());
  auto uncomp_data = decompress(compression, hbuffer);
  CUDF_CUDA_TRY(hipMemcpyAsync(buffer.data(),
                                reinterpret_cast<char*>(uncomp_data.data()),
                                uncomp_data.size() * sizeof(char),
                                hipMemcpyHostToDevice,
                                stream.value()));
  stream.synchronize();
  return buffer.first(uncomp_data.size());
}

size_type find_first_delimiter_in_chunk(host_span<std::unique_ptr<cudf::io::datasource>> sources,
                                        json_reader_options const& reader_opts,
                                        char const delimiter,
                                        rmm::cuda_stream_view stream)
{
  auto total_source_size = sources_size(sources, 0, 0) + (sources.size() - 1);
  rmm::device_uvector<char> buffer(total_source_size, stream);
  auto readbufspan = ingest_raw_input(buffer,
                                      sources,
                                      reader_opts.get_compression(),
                                      reader_opts.get_byte_range_offset(),
                                      reader_opts.get_byte_range_size(),
                                      stream);
  return find_first_delimiter(readbufspan, '\n', stream);
}

/**
 * @brief Get the byte range between record starts and ends starting from the given range.
 *
 * if get_byte_range_offset == 0, then we can skip the first delimiter search
 * if get_byte_range_offset != 0, then we need to search for the first delimiter in given range.
 * if not found, skip this chunk, if found, then search for first delimiter in next range until we
 * find a delimiter. Use this as actual range for parsing.
 *
 * @param sources Data sources to read from
 * @param reader_opts JSON reader options with range offset and range size
 * @param stream CUDA stream used for device memory operations and kernel launches
 * @returns Data source owning buffer enclosing the bytes read
 */
datasource::owning_buffer<rmm::device_uvector<char>> get_record_range_raw_input(
  host_span<std::unique_ptr<datasource>> sources,
  json_reader_options const& reader_opts,
  rmm::cuda_stream_view stream)
{
  CUDF_FUNC_RANGE();
  auto geometric_mean = [](double a, double b) { return std::sqrt(a * b); };

  size_t const total_source_size            = sources_size(sources, 0, 0);
  auto constexpr num_delimiter_chars        = 1;
  auto const num_extra_delimiters           = num_delimiter_chars * (sources.size() - 1);
  compression_type const reader_compression = reader_opts.get_compression();
  size_t const chunk_offset                 = reader_opts.get_byte_range_offset();
  size_t chunk_size                         = reader_opts.get_byte_range_size();

  CUDF_EXPECTS(total_source_size ? chunk_offset < total_source_size : !chunk_offset,
               "Invalid offsetting");
  auto should_load_all_sources = !chunk_size || chunk_size >= total_source_size - chunk_offset;
  chunk_size = should_load_all_sources ? total_source_size - chunk_offset : chunk_size;

  // Some magic numbers
  constexpr int num_subchunks               = 10;  // per chunk_size
  constexpr size_t min_subchunk_size        = 10000;
  int const num_subchunks_prealloced        = should_load_all_sources ? 0 : 3;
  constexpr int estimated_compression_ratio = 4;

  // NOTE: heuristic for choosing subchunk size: geometric mean of minimum subchunk size (set to
  // 10kb) and the byte range size

  size_t const size_per_subchunk =
    geometric_mean(std::ceil((double)chunk_size / num_subchunks), min_subchunk_size);

  // The allocation for single source compressed input is estimated by assuming a ~4:1
  // compression ratio. For uncompressed inputs, we can getter a better estimate using the idea
  // of subchunks.
  auto constexpr header_size = 4096;
  size_t const buffer_size =
    reader_compression != compression_type::NONE
      ? total_source_size * estimated_compression_ratio + header_size
      : std::min(total_source_size, chunk_size + num_subchunks_prealloced * size_per_subchunk) +
          num_extra_delimiters;
  rmm::device_uvector<char> buffer(buffer_size, stream);
  device_span<char> bufspan(buffer);

  // Offset within buffer indicating first read position
  std::int64_t buffer_offset = 0;
  auto readbufspan =
    ingest_raw_input(bufspan, sources, reader_compression, chunk_offset, chunk_size, stream);

  auto const shift_for_nonzero_offset = std::min<std::int64_t>(chunk_offset, 1);
  auto const first_delim_pos =
    chunk_offset == 0 ? 0 : find_first_delimiter(readbufspan, '\n', stream);
  if (first_delim_pos == -1) {
    // return empty owning datasource buffer
    auto empty_buf = rmm::device_uvector<char>(0, stream);
    return datasource::owning_buffer<rmm::device_uvector<char>>(std::move(empty_buf));
  } else if (!should_load_all_sources) {
    // Find next delimiter
    std::int64_t next_delim_pos = -1;
    size_t next_subchunk_start  = chunk_offset + chunk_size;
    while (next_subchunk_start < total_source_size && next_delim_pos < buffer_offset) {
      buffer_offset += readbufspan.size();
      readbufspan    = ingest_raw_input(bufspan.last(buffer_size - buffer_offset),
                                     sources,
                                     reader_compression,
                                     next_subchunk_start,
                                     size_per_subchunk,
                                     stream);
      next_delim_pos = find_first_delimiter(readbufspan, '\n', stream) + buffer_offset;
      if (next_delim_pos < buffer_offset) { next_subchunk_start += size_per_subchunk; }
    }
    if (next_delim_pos < buffer_offset) next_delim_pos = buffer_offset + readbufspan.size();

    return datasource::owning_buffer<rmm::device_uvector<char>>(
      std::move(buffer),
      reinterpret_cast<uint8_t*>(buffer.data()) + first_delim_pos + shift_for_nonzero_offset,
      next_delim_pos - first_delim_pos - shift_for_nonzero_offset);
  }
  return datasource::owning_buffer<rmm::device_uvector<char>>(
    std::move(buffer),
    reinterpret_cast<uint8_t*>(buffer.data()) + first_delim_pos + shift_for_nonzero_offset,
    readbufspan.size() - first_delim_pos - shift_for_nonzero_offset);
}

table_with_metadata read_json(host_span<std::unique_ptr<datasource>> sources,
                              json_reader_options const& reader_opts,
                              rmm::cuda_stream_view stream,
                              rmm::device_async_resource_ref mr)
{
  CUDF_FUNC_RANGE();

  // TODO remove this if-statement once legacy is removed
#pragma GCC diagnostic push
#pragma GCC diagnostic ignored "-Wdeprecated-declarations"
  if (reader_opts.is_enabled_legacy()) {
    return legacy::read_json(sources, reader_opts, stream, mr);
  }
#pragma GCC diagnostic pop

  if (reader_opts.get_byte_range_offset() != 0 or reader_opts.get_byte_range_size() != 0) {
    CUDF_EXPECTS(reader_opts.is_enabled_lines(),
                 "Specifying a byte range is supported only for JSON Lines");
  }

  if (sources.size() > 1) {
    CUDF_EXPECTS(reader_opts.get_compression() == compression_type::NONE,
                 "Multiple compressed inputs are not supported");
    CUDF_EXPECTS(reader_opts.is_enabled_lines(),
                 "Multiple inputs are supported only for JSON Lines format");
  }

  datasource::owning_buffer<rmm::device_uvector<char>> bufview =
    get_record_range_raw_input(sources, reader_opts, stream);

  // If input JSON buffer has single quotes and option to normalize single quotes is enabled,
  // invoke pre-processing FST
  if (reader_opts.is_enabled_normalize_single_quotes()) {
    normalize_single_quotes(bufview, stream, rmm::mr::get_current_device_resource());
  }

  // If input JSON buffer has unquoted spaces and tabs and option to normalize whitespaces is
  // enabled, invoke pre-processing FST
  if (reader_opts.is_enabled_normalize_whitespace()) {
    normalize_whitespace(bufview, stream, rmm::mr::get_current_device_resource());
  }

  auto buffer =
    cudf::device_span<char const>(reinterpret_cast<char const*>(bufview.data()), bufview.size());
  stream.synchronize();
  return device_parse_nested_json(buffer, reader_opts, stream, mr);
}

}  // namespace cudf::io::json::detail
