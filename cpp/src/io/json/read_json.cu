#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2022-2024, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "io/comp/io_uncomp.hpp"
#include "io/json/nested_json.hpp"
#include "read_json.hpp"

#include <cudf/concatenate.hpp>
#include <cudf/detail/nvtx/ranges.hpp>
#include <cudf/detail/utilities/integer_utils.hpp>
#include <cudf/detail/utilities/stream_pool.hpp>
#include <cudf/detail/utilities/vector_factories.hpp>
#include <cudf/io/detail/json.hpp>
#include <cudf/utilities/error.hpp>
#include <cudf/utilities/span.hpp>

#include <rmm/device_uvector.hpp>
#include <rmm/exec_policy.hpp>
#include <rmm/resource_ref.hpp>

#include <thrust/iterator/constant_iterator.h>
#include <thrust/scatter.h>

#include <numeric>

namespace cudf::io::json::detail {

size_t sources_size(host_span<std::unique_ptr<datasource>> const sources,
                    size_t range_offset,
                    size_t range_size)
{
  return std::accumulate(sources.begin(), sources.end(), 0ul, [=](size_t sum, auto& source) {
    auto const size = source->size();
    // TODO take care of 0, 0, or *, 0 case.
    return sum +
           (range_size == 0 or range_offset + range_size > size ? size - range_offset : range_size);
  });
}

/**
 * @brief Read from array of data sources into RMM buffer. The size of the returned device span
          can be larger than the number of bytes requested from the list of sources when
          the range to be read spans across multiple sources. This is due to the delimiter
          characters inserted after the end of each accessed source.
 *
 * @param buffer Device span buffer to which data is read
 * @param sources Array of data sources
 * @param compression Compression format of source
 * @param range_offset Number of bytes to skip from source start
 * @param range_size Number of bytes to read from source
 * @param stream CUDA stream used for device memory operations and kernel launches
 * @returns A subspan of the input device span containing data read
 */
device_span<char> ingest_raw_input(device_span<char> buffer,
                                   host_span<std::unique_ptr<datasource>> sources,
                                   compression_type compression,
                                   size_t range_offset,
                                   size_t range_size,
                                   rmm::cuda_stream_view stream)
{
  CUDF_FUNC_RANGE();
  // We append a line delimiter between two files to make sure the last line of file i and the first
  // line of file i+1 don't end up on the same JSON line, if file i does not already end with a line
  // delimiter.
  auto constexpr num_delimiter_chars = 1;

  if (compression == compression_type::NONE) {
    std::vector<size_t> delimiter_map{};
    std::vector<size_t> prefsum_source_sizes(sources.size());
    std::vector<std::unique_ptr<datasource::buffer>> h_buffers;
    delimiter_map.reserve(sources.size());
    size_t bytes_read = 0;
    std::transform_inclusive_scan(sources.begin(),
                                  sources.end(),
                                  prefsum_source_sizes.begin(),
                                  std::plus<size_t>{},
                                  [](std::unique_ptr<datasource> const& s) { return s->size(); });
    auto upper =
      std::upper_bound(prefsum_source_sizes.begin(), prefsum_source_sizes.end(), range_offset);
    size_t start_source = std::distance(prefsum_source_sizes.begin(), upper);

    auto const total_bytes_to_read =
      std::min(range_size, prefsum_source_sizes.back() - range_offset);
    range_offset -= start_source ? prefsum_source_sizes[start_source - 1] : 0;
    for (size_t i = start_source; i < sources.size() && bytes_read < total_bytes_to_read; i++) {
      if (sources[i]->is_empty()) continue;
      auto data_size =
        std::min(sources[i]->size() - range_offset, total_bytes_to_read - bytes_read);
      auto destination = reinterpret_cast<uint8_t*>(buffer.data()) + bytes_read +
                         (num_delimiter_chars * delimiter_map.size());
      if (sources[i]->is_device_read_preferred(data_size)) {
        bytes_read += sources[i]->device_read(range_offset, data_size, destination, stream);
      } else {
        h_buffers.emplace_back(sources[i]->host_read(range_offset, data_size));
        auto const& h_buffer = h_buffers.back();
        CUDF_CUDA_TRY(hipMemcpyAsync(
          destination, h_buffer->data(), h_buffer->size(), hipMemcpyHostToDevice, stream.value()));
        bytes_read += h_buffer->size();
      }
      range_offset = 0;
      delimiter_map.push_back(bytes_read + (num_delimiter_chars * delimiter_map.size()));
    }
    // Removing delimiter inserted after last non-empty source is read
    if (!delimiter_map.empty()) { delimiter_map.pop_back(); }

    // If this is a multi-file source, we scatter the JSON line delimiters between files
    if (sources.size() > 1) {
      static_assert(num_delimiter_chars == 1,
                    "Currently only single-character delimiters are supported");
      auto const delimiter_source = thrust::make_constant_iterator('\n');
      auto const d_delimiter_map  = cudf::detail::make_device_uvector_async(
        delimiter_map, stream, rmm::mr::get_current_device_resource());
      thrust::scatter(rmm::exec_policy_nosync(stream),
                      delimiter_source,
                      delimiter_source + d_delimiter_map.size(),
                      d_delimiter_map.data(),
                      buffer.data());
    }
    stream.synchronize();
    return buffer.first(bytes_read + (delimiter_map.size() * num_delimiter_chars));
  }
  // TODO: allow byte range reading from multiple compressed files.
  auto remaining_bytes_to_read = std::min(range_size, sources[0]->size() - range_offset);
  auto hbuffer                 = std::vector<uint8_t>(remaining_bytes_to_read);
  // Single read because only a single compressed source is supported
  // Reading to host because decompression of a single block is much faster on the CPU
  sources[0]->host_read(range_offset, remaining_bytes_to_read, hbuffer.data());
  auto uncomp_data = decompress(compression, hbuffer);
  CUDF_CUDA_TRY(hipMemcpyAsync(buffer.data(),
                                reinterpret_cast<char*>(uncomp_data.data()),
                                uncomp_data.size() * sizeof(char),
                                hipMemcpyHostToDevice,
                                stream.value()));
  stream.synchronize();
  return buffer.first(uncomp_data.size());
}

size_type find_first_delimiter_in_chunk(host_span<std::unique_ptr<cudf::io::datasource>> sources,
                                        json_reader_options const& reader_opts,
                                        char const delimiter,
                                        rmm::cuda_stream_view stream)
{
  auto total_source_size = sources_size(sources, 0, 0) + (sources.size() - 1);
  rmm::device_uvector<char> buffer(total_source_size, stream);
  auto readbufspan = ingest_raw_input(buffer,
                                      sources,
                                      reader_opts.get_compression(),
                                      reader_opts.get_byte_range_offset(),
                                      reader_opts.get_byte_range_size(),
                                      stream);
  return find_first_delimiter(readbufspan, '\n', stream);
}

/**
 * @brief Get the byte range between record starts and ends starting from the given range.
 *
 * if get_byte_range_offset == 0, then we can skip the first delimiter search
 * if get_byte_range_offset != 0, then we need to search for the first delimiter in given range.
 * if not found, skip this chunk, if found, then search for first delimiter in next range until we
 * find a delimiter. Use this as actual range for parsing.
 *
 * @param sources Data sources to read from
 * @param reader_opts JSON reader options with range offset and range size
 * @param stream CUDA stream used for device memory operations and kernel launches
 * @returns Data source owning buffer enclosing the bytes read
 */
datasource::owning_buffer<rmm::device_uvector<char>> get_record_range_raw_input(
  host_span<std::unique_ptr<datasource>> sources,
  json_reader_options const& reader_opts,
  rmm::cuda_stream_view stream)
{
  CUDF_FUNC_RANGE();
  auto geometric_mean = [](double a, double b) { return std::sqrt(a * b); };

  size_t const total_source_size            = sources_size(sources, 0, 0);
  auto constexpr num_delimiter_chars        = 1;
  auto const num_extra_delimiters           = num_delimiter_chars * (sources.size() - 1);
  compression_type const reader_compression = reader_opts.get_compression();
  size_t const chunk_offset                 = reader_opts.get_byte_range_offset();
  size_t chunk_size                         = reader_opts.get_byte_range_size();

  CUDF_EXPECTS(total_source_size ? chunk_offset < total_source_size : !chunk_offset,
               "Invalid offsetting",
               std::invalid_argument);
  auto should_load_all_sources = !chunk_size || chunk_size >= total_source_size - chunk_offset;
  chunk_size = should_load_all_sources ? total_source_size - chunk_offset : chunk_size;

  // Some magic numbers
  constexpr int num_subchunks               = 10;  // per chunk_size
  constexpr size_t min_subchunk_size        = 10000;
  int const num_subchunks_prealloced        = should_load_all_sources ? 0 : 3;
  constexpr int estimated_compression_ratio = 4;

  // NOTE: heuristic for choosing subchunk size: geometric mean of minimum subchunk size (set to
  // 10kb) and the byte range size

  size_t const size_per_subchunk =
    geometric_mean(std::ceil((double)chunk_size / num_subchunks), min_subchunk_size);

  // The allocation for single source compressed input is estimated by assuming a ~4:1
  // compression ratio. For uncompressed inputs, we can getter a better estimate using the idea
  // of subchunks.
  auto constexpr header_size = 4096;
  size_t const buffer_size =
    reader_compression != compression_type::NONE
      ? total_source_size * estimated_compression_ratio + header_size
      : std::min(total_source_size, chunk_size + num_subchunks_prealloced * size_per_subchunk) +
          num_extra_delimiters;
  rmm::device_uvector<char> buffer(buffer_size, stream);
  device_span<char> bufspan(buffer);

  // Offset within buffer indicating first read position
  std::int64_t buffer_offset = 0;
  auto readbufspan =
    ingest_raw_input(bufspan, sources, reader_compression, chunk_offset, chunk_size, stream);

  auto const shift_for_nonzero_offset = std::min<std::int64_t>(chunk_offset, 1);
  auto const first_delim_pos =
    chunk_offset == 0 ? 0 : find_first_delimiter(readbufspan, '\n', stream);
  if (first_delim_pos == -1) {
    // return empty owning datasource buffer
    auto empty_buf = rmm::device_uvector<char>(0, stream);
    return datasource::owning_buffer<rmm::device_uvector<char>>(std::move(empty_buf));
  } else if (!should_load_all_sources) {
    // Find next delimiter
    std::int64_t next_delim_pos = -1;
    size_t next_subchunk_start  = chunk_offset + chunk_size;
    while (next_subchunk_start < total_source_size && next_delim_pos < buffer_offset) {
      buffer_offset += readbufspan.size();
      readbufspan    = ingest_raw_input(bufspan.last(buffer_size - buffer_offset),
                                     sources,
                                     reader_compression,
                                     next_subchunk_start,
                                     size_per_subchunk,
                                     stream);
      next_delim_pos = find_first_delimiter(readbufspan, '\n', stream) + buffer_offset;
      if (next_delim_pos < buffer_offset) { next_subchunk_start += size_per_subchunk; }
    }
    if (next_delim_pos < buffer_offset) next_delim_pos = buffer_offset + readbufspan.size();

    return datasource::owning_buffer<rmm::device_uvector<char>>(
      std::move(buffer),
      reinterpret_cast<uint8_t*>(buffer.data()) + first_delim_pos + shift_for_nonzero_offset,
      next_delim_pos - first_delim_pos - shift_for_nonzero_offset);
  }
  return datasource::owning_buffer<rmm::device_uvector<char>>(
    std::move(buffer),
    reinterpret_cast<uint8_t*>(buffer.data()) + first_delim_pos + shift_for_nonzero_offset,
    readbufspan.size() - first_delim_pos - shift_for_nonzero_offset);
}

table_with_metadata read_batch(host_span<std::unique_ptr<datasource>> sources,
                               json_reader_options const& reader_opts,
                               rmm::cuda_stream_view stream,
                               rmm::device_async_resource_ref mr)
{
  CUDF_FUNC_RANGE();
  datasource::owning_buffer<rmm::device_uvector<char>> bufview =
    get_record_range_raw_input(sources, reader_opts, stream);

  // If input JSON buffer has single quotes and option to normalize single quotes is enabled,
  // invoke pre-processing FST
  if (reader_opts.is_enabled_normalize_single_quotes()) {
    normalize_single_quotes(bufview, stream, rmm::mr::get_current_device_resource());
  }

  // If input JSON buffer has unquoted spaces and tabs and option to normalize whitespaces is
  // enabled, invoke pre-processing FST
  if (reader_opts.is_enabled_normalize_whitespace()) {
    normalize_whitespace(bufview, stream, rmm::mr::get_current_device_resource());
  }

  auto buffer =
    cudf::device_span<char const>(reinterpret_cast<char const*>(bufview.data()), bufview.size());
  stream.synchronize();
  return device_parse_nested_json(buffer, reader_opts, stream, mr);
}

table_with_metadata read_json(host_span<std::unique_ptr<datasource>> sources,
                              json_reader_options const& reader_opts,
                              rmm::cuda_stream_view stream,
                              rmm::device_async_resource_ref mr)
{
  CUDF_FUNC_RANGE();

  if (reader_opts.get_byte_range_offset() != 0 or reader_opts.get_byte_range_size() != 0) {
    CUDF_EXPECTS(reader_opts.is_enabled_lines(),
                 "Specifying a byte range is supported only for JSON Lines");
  }

  if (sources.size() > 1) {
    CUDF_EXPECTS(reader_opts.get_compression() == compression_type::NONE,
                 "Multiple compressed inputs are not supported");
    CUDF_EXPECTS(reader_opts.is_enabled_lines(),
                 "Multiple inputs are supported only for JSON Lines format");
  }

  std::for_each(sources.begin(), sources.end(), [](auto const& source) {
    CUDF_EXPECTS(source->size() < std::numeric_limits<int>::max(),
                 "The size of each source file must be less than INT_MAX bytes");
  });

  constexpr size_t batch_size_ub = std::numeric_limits<int>::max();
  size_t const chunk_offset      = reader_opts.get_byte_range_offset();
  size_t chunk_size              = reader_opts.get_byte_range_size();
  chunk_size                     = !chunk_size ? sources_size(sources, 0, 0) : chunk_size;

  // Identify the position of starting source file from which to begin batching based on
  // byte range offset. If the offset is larger than the sum of all source
  // sizes, then start_source is total number of source files i.e. no file is read
  size_t const start_source = [&]() {
    size_t sum = 0;
    for (size_t src_idx = 0; src_idx < sources.size(); ++src_idx) {
      if (sum + sources[src_idx]->size() > chunk_offset) return src_idx;
      sum += sources[src_idx]->size();
    }
    return sources.size();
  }();

  // Construct batches of source files, with starting position of batches indicated by
  // batch_positions. The size of each batch i.e. the sum of sizes of the source files in the batch
  // is capped at INT_MAX bytes.
  size_t cur_size = 0;
  std::vector<size_t> batch_positions;
  std::vector<size_t> batch_sizes;
  batch_positions.push_back(0);
  for (size_t i = start_source; i < sources.size(); i++) {
    cur_size += sources[i]->size();
    if (cur_size >= batch_size_ub) {
      batch_positions.push_back(i);
      batch_sizes.push_back(cur_size - sources[i]->size());
      cur_size = sources[i]->size();
    }
  }
  batch_positions.push_back(sources.size());
  batch_sizes.push_back(cur_size);

  // If there is a single batch, then we can directly return the table without the
  // unnecessary concatenate
  if (batch_sizes.size() == 1) return read_batch(sources, reader_opts, stream, mr);

  std::vector<cudf::io::table_with_metadata> partial_tables;
  json_reader_options batched_reader_opts{reader_opts};

  // Dispatch individual batches to read_batch and push the resulting table into
  // partial_tables array. Note that the reader options need to be updated for each
  // batch to adjust byte range offset and byte range size.
  for (size_t i = 0; i < batch_sizes.size(); i++) {
    batched_reader_opts.set_byte_range_size(std::min(batch_sizes[i], chunk_size));
    partial_tables.emplace_back(read_batch(
      host_span<std::unique_ptr<datasource>>(sources.begin() + batch_positions[i],
                                             batch_positions[i + 1] - batch_positions[i]),
      batched_reader_opts,
      stream,
      rmm::mr::get_current_device_resource()));
    if (chunk_size <= batch_sizes[i]) break;
    chunk_size -= batch_sizes[i];
    batched_reader_opts.set_byte_range_offset(0);
  }

  auto expects_schema_equality =
    std::all_of(partial_tables.begin() + 1,
                partial_tables.end(),
                [&gt = partial_tables[0].metadata.schema_info](auto& ptbl) {
                  return ptbl.metadata.schema_info == gt;
                });
  CUDF_EXPECTS(expects_schema_equality,
               "Mismatch in JSON schema across batches in multi-source multi-batch reading");

  auto partial_table_views = std::vector<cudf::table_view>(partial_tables.size());
  std::transform(partial_tables.begin(),
                 partial_tables.end(),
                 partial_table_views.begin(),
                 [](auto const& table) { return table.tbl->view(); });
  return table_with_metadata{cudf::concatenate(partial_table_views, stream, mr),
                             {partial_tables[0].metadata.schema_info}};
}

}  // namespace cudf::io::json::detail
