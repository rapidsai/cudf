#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2022-2024, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "io/utilities/parsing_utils.cuh"
#include "io/utilities/string_parsing.hpp"
#include "nested_json.hpp"

#include <cudf/column/column_factories.hpp>
#include <cudf/detail/null_mask.hpp>
#include <cudf/detail/nvtx/ranges.hpp>
#include <cudf/detail/utilities/vector_factories.hpp>
#include <cudf/detail/utilities/visitor_overload.hpp>
#include <cudf/io/detail/json.hpp>
#include <cudf/types.hpp>
#include <cudf/utilities/error.hpp>
#include <cudf/utilities/memory_resource.hpp>
#include <cudf/utilities/span.hpp>

#include <rmm/cuda_stream_view.hpp>
#include <rmm/device_uvector.hpp>
#include <rmm/exec_policy.hpp>

#include <cuda/atomic>
#include <cuda/functional>
#include <thrust/for_each.h>
#include <thrust/functional.h>
#include <thrust/gather.h>
#include <thrust/iterator/discard_iterator.h>
#include <thrust/iterator/permutation_iterator.h>
#include <thrust/iterator/zip_iterator.h>
#include <thrust/reduce.h>
#include <thrust/transform.h>
#include <thrust/unique.h>

namespace cudf::io::json::detail {

auto to_cat = [](auto v) -> std::string {
  switch (v) {
    case NC_STRUCT: return " S";
    case NC_LIST: return " L";
    case NC_STR: return " \"";
    case NC_VAL: return " V";
    case NC_FN: return " F";
    case NC_ERR: return "ER";
    default: return "UN";
  };
};
auto to_int    = [](auto v) { return std::to_string(static_cast<int>(v)); };
auto print_vec = [](auto const& cpu, auto const name, auto converter) {
  for (auto const& v : cpu)
    printf("%3s,", converter(v).c_str());
  std::cout << name << std::endl;
};

void print_tree(host_span<SymbolT const> input,
                tree_meta_t const& d_gpu_tree,
                rmm::cuda_stream_view stream)
{
  print_vec(cudf::detail::make_host_vector_sync(d_gpu_tree.node_categories, stream),
            "node_categories",
            to_cat);
  print_vec(cudf::detail::make_host_vector_sync(d_gpu_tree.parent_node_ids, stream),
            "parent_node_ids",
            to_int);
  print_vec(
    cudf::detail::make_host_vector_sync(d_gpu_tree.node_levels, stream), "node_levels", to_int);
  auto node_range_begin = cudf::detail::make_host_vector_sync(d_gpu_tree.node_range_begin, stream);
  auto node_range_end   = cudf::detail::make_host_vector_sync(d_gpu_tree.node_range_end, stream);
  print_vec(node_range_begin, "node_range_begin", to_int);
  print_vec(node_range_end, "node_range_end", to_int);
  for (int i = 0; i < int(node_range_begin.size()); i++) {
    printf("%3s ",
           std::string(input.data() + node_range_begin[i], node_range_end[i] - node_range_begin[i])
             .c_str());
  }
  printf(" (JSON)\n");
}

/**
 * @brief Reduces node tree representation to column tree representation.
 *
 * @param tree Node tree representation of JSON string
 * @param original_col_ids Column ids of nodes
 * @param sorted_col_ids Sorted column ids of nodes
 * @param ordered_node_ids Node ids of nodes sorted by column ids
 * @param row_offsets Row offsets of nodes
 * @param is_array_of_arrays Whether the tree is an array of arrays
 * @param row_array_parent_col_id Column id of row array, if is_array_of_arrays is true
 * @param stream CUDA stream used for device memory operations and kernel launches
 * @return A tuple of column tree representation of JSON string, column ids of columns, and
 * max row offsets of columns
 */
std::tuple<tree_meta_t, rmm::device_uvector<NodeIndexT>, rmm::device_uvector<size_type>>
reduce_to_column_tree(tree_meta_t& tree,
                      device_span<NodeIndexT const> original_col_ids,
                      device_span<NodeIndexT const> sorted_col_ids,
                      device_span<NodeIndexT const> ordered_node_ids,
                      device_span<size_type const> row_offsets,
                      bool is_array_of_arrays,
                      NodeIndexT const row_array_parent_col_id,
                      rmm::cuda_stream_view stream)
{
  CUDF_FUNC_RANGE();

  // 1. column count for allocation
  auto const num_columns = thrust::unique_count(
    rmm::exec_policy_nosync(stream), sorted_col_ids.begin(), sorted_col_ids.end());

  // 2. reduce_by_key {col_id}, {row_offset}, max.
  rmm::device_uvector<NodeIndexT> unique_col_ids(num_columns, stream);
  rmm::device_uvector<size_type> max_row_offsets(num_columns, stream);
  auto ordered_row_offsets =
    thrust::make_permutation_iterator(row_offsets.begin(), ordered_node_ids.begin());
  thrust::reduce_by_key(rmm::exec_policy(stream),
                        sorted_col_ids.begin(),
                        sorted_col_ids.end(),
                        ordered_row_offsets,
                        unique_col_ids.begin(),
                        max_row_offsets.begin(),
                        thrust::equal_to<size_type>(),
                        thrust::maximum<size_type>());

  // 3. reduce_by_key {col_id}, {node_categories} - custom opp (*+v=*, v+v=v, *+#=E)
  rmm::device_uvector<NodeT> column_categories(num_columns, stream);
  thrust::reduce_by_key(
    rmm::exec_policy(stream),
    sorted_col_ids.begin(),
    sorted_col_ids.end(),
    thrust::make_permutation_iterator(tree.node_categories.begin(), ordered_node_ids.begin()),
    unique_col_ids.begin(),
    column_categories.begin(),
    thrust::equal_to<size_type>(),
    [] __device__(NodeT type_a, NodeT type_b) -> NodeT {
      auto is_a_leaf = (type_a == NC_VAL || type_a == NC_STR);
      auto is_b_leaf = (type_b == NC_VAL || type_b == NC_STR);
      // (v+v=v, *+*=*,  *+v=*, *+#=E, NESTED+VAL=NESTED)
      // *+*=*, v+v=v
      if (type_a == type_b) {
        return type_a;
      } else if (is_a_leaf) {
        // *+v=*, N+V=N
        // STRUCT/LIST + STR/VAL = STRUCT/LIST, STR/VAL + FN = ERR, STR/VAL + STR = STR
        return type_b == NC_FN ? NC_ERR : (is_b_leaf ? NC_STR : type_b);
      } else if (is_b_leaf) {
        return type_a == NC_FN ? NC_ERR : (is_a_leaf ? NC_STR : type_a);
      }
      // *+#=E
      return NC_ERR;
    });

  // 4. unique_copy parent_node_ids, ranges
  rmm::device_uvector<TreeDepthT> column_levels(num_columns, stream);  // not required
  rmm::device_uvector<NodeIndexT> parent_col_ids(num_columns, stream);
  rmm::device_uvector<SymbolOffsetT> col_range_begin(num_columns, stream);  // Field names
  rmm::device_uvector<SymbolOffsetT> col_range_end(num_columns, stream);
  rmm::device_uvector<size_type> unique_node_ids(num_columns, stream);
  thrust::unique_by_key_copy(rmm::exec_policy_nosync(stream),
                             sorted_col_ids.begin(),
                             sorted_col_ids.end(),
                             ordered_node_ids.begin(),
                             thrust::make_discard_iterator(),
                             unique_node_ids.begin());

  thrust::copy_n(
    rmm::exec_policy_nosync(stream),
    thrust::make_zip_iterator(
      thrust::make_permutation_iterator(tree.node_levels.begin(), unique_node_ids.begin()),
      thrust::make_permutation_iterator(tree.parent_node_ids.begin(), unique_node_ids.begin()),
      thrust::make_permutation_iterator(tree.node_range_begin.begin(), unique_node_ids.begin()),
      thrust::make_permutation_iterator(tree.node_range_end.begin(), unique_node_ids.begin())),
    unique_node_ids.size(),
    thrust::make_zip_iterator(column_levels.begin(),
                              parent_col_ids.begin(),
                              col_range_begin.begin(),
                              col_range_end.begin()));

  // convert parent_node_ids to parent_col_ids
  thrust::transform(
    rmm::exec_policy_nosync(stream),
    parent_col_ids.begin(),
    parent_col_ids.end(),
    parent_col_ids.begin(),
    [col_ids = original_col_ids.begin()] __device__(auto parent_node_id) -> size_type {
      return parent_node_id == parent_node_sentinel ? parent_node_sentinel
                                                    : col_ids[parent_node_id];
    });

  // condition is true if parent is not a list, or sentinel/root
  // Special case to return true if parent is a list and is_array_of_arrays is true
  auto is_non_list_parent = [column_categories = column_categories.begin(),
                             is_array_of_arrays,
                             row_array_parent_col_id] __device__(auto parent_col_id) -> bool {
    return !(parent_col_id == parent_node_sentinel ||
             column_categories[parent_col_id] == NC_LIST &&
               (!is_array_of_arrays || parent_col_id != row_array_parent_col_id));
  };

  // Mixed types in List children go to different columns,
  // so all immediate children of list column should have same max_row_offsets.
  //   create list's children max_row_offsets array. (initialize to zero)
  //   atomicMax on  children max_row_offsets array.
  //   gather the max_row_offsets from children row offset array.
  {
    auto list_parents_children_max_row_offsets =
      cudf::detail::make_zeroed_device_uvector_async<NodeIndexT>(
        static_cast<std::size_t>(num_columns), stream, cudf::get_current_device_resource_ref());
    thrust::for_each(rmm::exec_policy_nosync(stream),
                     unique_col_ids.begin(),
                     unique_col_ids.end(),
                     [column_categories = column_categories.begin(),
                      parent_col_ids    = parent_col_ids.begin(),
                      max_row_offsets   = max_row_offsets.begin(),
                      list_parents_children_max_row_offsets =
                        list_parents_children_max_row_offsets.begin()] __device__(auto col_id) {
                       auto parent_col_id = parent_col_ids[col_id];
                       if (parent_col_id != parent_node_sentinel and
                           column_categories[parent_col_id] == node_t::NC_LIST) {
                         cuda::atomic_ref<NodeIndexT, cuda::thread_scope_device> ref{
                           *(list_parents_children_max_row_offsets + parent_col_id)};
                         ref.fetch_max(max_row_offsets[col_id], cuda::std::memory_order_relaxed);
                       }
                     });

    thrust::gather_if(
      rmm::exec_policy_nosync(stream),
      parent_col_ids.begin(),
      parent_col_ids.end(),
      parent_col_ids.begin(),
      list_parents_children_max_row_offsets.begin(),
      max_row_offsets.begin(),
      [column_categories = column_categories.begin()] __device__(size_type parent_col_id) {
        return parent_col_id != parent_node_sentinel and
               column_categories[parent_col_id] == node_t::NC_LIST;
      });
  }

  // copy lists' max_row_offsets to children.
  // all structs should have same size.
  thrust::transform_if(
    rmm::exec_policy_nosync(stream),
    unique_col_ids.begin(),
    unique_col_ids.end(),
    max_row_offsets.begin(),
    [column_categories = column_categories.begin(),
     is_non_list_parent,
     parent_col_ids  = parent_col_ids.begin(),
     max_row_offsets = max_row_offsets.begin()] __device__(size_type col_id) {
      auto parent_col_id = parent_col_ids[col_id];
      // condition is true if parent is not a list, or sentinel/root
      while (is_non_list_parent(parent_col_id)) {
        col_id        = parent_col_id;
        parent_col_id = parent_col_ids[parent_col_id];
      }
      return max_row_offsets[col_id];
    },
    [column_categories = column_categories.begin(),
     is_non_list_parent,
     parent_col_ids = parent_col_ids.begin()] __device__(size_type col_id) {
      auto parent_col_id = parent_col_ids[col_id];
      // condition is true if parent is not a list, or sentinel/root
      return is_non_list_parent(parent_col_id);
    });

  // For Struct and List (to avoid copying entire strings when mixed type as string is enabled)
  thrust::transform_if(
    rmm::exec_policy_nosync(stream),
    col_range_begin.begin(),
    col_range_begin.end(),
    column_categories.begin(),
    col_range_end.begin(),
    [] __device__(auto i) { return i + 1; },
    [] __device__(NodeT type) { return type == NC_STRUCT || type == NC_LIST; });

  return std::tuple{tree_meta_t{std::move(column_categories),
                                std::move(parent_col_ids),
                                std::move(column_levels),
                                std::move(col_range_begin),
                                std::move(col_range_end)},
                    std::move(unique_col_ids),
                    std::move(max_row_offsets)};
}

std::pair<std::unique_ptr<column>, std::vector<column_name_info>> device_json_column_to_cudf_column(
  device_json_column& json_col,
  device_span<SymbolT const> d_input,
  cudf::io::parse_options const& options,
  bool prune_columns,
  std::optional<schema_element> schema,
  rmm::cuda_stream_view stream,
  rmm::device_async_resource_ref mr)
{
  CUDF_FUNC_RANGE();
  auto validity_size_check = [](device_json_column& json_col) {
    CUDF_EXPECTS(json_col.validity.size() >= bitmask_allocation_size_bytes(json_col.num_rows),
                 "valid_count is too small");
  };
  auto make_validity = [stream, validity_size_check](
                         device_json_column& json_col) -> std::pair<rmm::device_buffer, size_type> {
    validity_size_check(json_col);
    auto null_count = cudf::detail::null_count(
      static_cast<bitmask_type*>(json_col.validity.data()), 0, json_col.num_rows, stream);
    // full null_mask is always required for parse_data
    return {std::move(json_col.validity), null_count};
    // Note: json_col modified here, moves this memory
  };

  auto get_child_schema = [schema](auto child_name) -> std::optional<schema_element> {
    if (schema.has_value()) {
      auto const result = schema.value().child_types.find(child_name);
      if (result != std::end(schema.value().child_types)) { return result->second; }
    }
    return {};
  };

  switch (json_col.type) {
    case json_col_t::StringColumn: {
      // move string_offsets to GPU and transform to string column
      auto const col_size      = json_col.string_offsets.size();
      using char_length_pair_t = thrust::pair<char const*, size_type>;
      CUDF_EXPECTS(json_col.string_offsets.size() == json_col.string_lengths.size(),
                   "string offset, string length mismatch");
      rmm::device_uvector<char_length_pair_t> d_string_data(col_size, stream);
      // TODO how about directly storing pair<char*, size_t> in json_column?

      auto [result_bitmask, null_count] = make_validity(json_col);

      data_type target_type{};
      std::unique_ptr<column> col{};
      if (options.normalize_whitespace && json_col.forced_as_string_column) {
        CUDF_EXPECTS(prune_columns || options.mixed_types_as_string,
                     "Whitespace normalization of nested columns requested as string requires "
                     "either prune_columns or mixed_types_as_string to be enabled");
        auto [normalized_d_input, col_offsets, col_lengths] =
          cudf::io::json::detail::normalize_whitespace(
            d_input, json_col.string_offsets, json_col.string_lengths, stream, mr);
        auto offset_length_it = thrust::make_zip_iterator(col_offsets.begin(), col_lengths.begin());
        target_type           = data_type{type_id::STRING};
        // Convert strings to the inferred data type
        col = parse_data(normalized_d_input.data(),
                         offset_length_it,
                         col_size,
                         target_type,
                         std::move(result_bitmask),
                         null_count,
                         options.view(),
                         stream,
                         mr);
      } else {
        auto offset_length_it = thrust::make_zip_iterator(json_col.string_offsets.begin(),
                                                          json_col.string_lengths.begin());
        if (schema.has_value()) {
#ifdef NJP_DEBUG_PRINT
          std::cout << "-> explicit type: "
                    << (schema.has_value() ? std::to_string(static_cast<int>(schema->type.id()))
                                           : "n/a");
#endif
          target_type = schema.value().type;
        }
        // Infer column type, if we don't have an explicit type for it
        else {
          target_type = cudf::io::detail::infer_data_type(
            options.json_view(), d_input, offset_length_it, col_size, stream);
        }
        // Convert strings to the inferred data type
        col = parse_data(d_input.data(),
                         offset_length_it,
                         col_size,
                         target_type,
                         std::move(result_bitmask),
                         null_count,
                         options.view(),
                         stream,
                         mr);
      }

      // Reset nullable if we do not have nulls
      // This is to match the existing JSON reader's behaviour:
      // - Non-string columns will always be returned as nullable
      // - String columns will be returned as nullable, iff there's at least one null entry
      if (col->null_count() == 0) { col->set_null_mask(rmm::device_buffer{0, stream, mr}, 0); }

      // For string columns return ["offsets", "char"] schema
      if (target_type.id() == type_id::STRING) {
        return {std::move(col), std::vector<column_name_info>{{"offsets"}, {"chars"}}};
      }
      // Non-string leaf-columns (e.g., numeric) do not have child columns in the schema
      return {std::move(col), std::vector<column_name_info>{}};
    }
    case json_col_t::StructColumn: {
      std::vector<std::unique_ptr<column>> child_columns;
      std::vector<column_name_info> column_names{};
      size_type num_rows{json_col.num_rows};
      // Create children columns
      for (auto const& col_name : json_col.column_order) {
        auto const& col = json_col.child_columns.find(col_name);
        column_names.emplace_back(col->first);
        auto& child_col           = col->second;
        auto child_schema_element = get_child_schema(col_name);
        if (!prune_columns or child_schema_element.has_value()) {
          auto [child_column, names] = device_json_column_to_cudf_column(
            child_col, d_input, options, prune_columns, child_schema_element, stream, mr);
          CUDF_EXPECTS(num_rows == child_column->size(),
                       "All children columns must have the same size");
          child_columns.push_back(std::move(child_column));
          column_names.back().children = names;
        }
      }
      auto [result_bitmask, null_count] = make_validity(json_col);
      // The null_mask is set after creation of struct column is to skip the superimpose_nulls and
      // null validation applied in make_structs_column factory, which is not needed for json
      auto ret_col = make_structs_column(num_rows, std::move(child_columns), 0, {}, stream, mr);
      if (null_count != 0) { ret_col->set_null_mask(std::move(result_bitmask), null_count); }
      return {std::move(ret_col), column_names};
    }
    case json_col_t::ListColumn: {
      size_type num_rows = json_col.child_offsets.size() - 1;
      std::vector<column_name_info> column_names{};
      column_names.emplace_back("offsets");
      column_names.emplace_back(
        json_col.child_columns.empty() ? list_child_name : json_col.child_columns.begin()->first);

      // Note: json_col modified here, reuse the memory
      auto offsets_column = std::make_unique<column>(data_type{type_id::INT32},
                                                     num_rows + 1,
                                                     json_col.child_offsets.release(),
                                                     rmm::device_buffer{},
                                                     0);
      // Create children column
      auto child_schema_element = json_col.child_columns.empty()
                                    ? std::optional<schema_element>{}
                                    : get_child_schema(json_col.child_columns.begin()->first);
      auto [child_column, names] =
        json_col.child_columns.empty() or (prune_columns and !child_schema_element.has_value())
          ? std::pair<std::unique_ptr<column>,
                      // EMPTY type could not used because gather throws exception on EMPTY type.
                      std::vector<column_name_info>>{std::make_unique<column>(
                                                       data_type{type_id::INT8},
                                                       0,
                                                       rmm::device_buffer{},
                                                       rmm::device_buffer{},
                                                       0),
                                                     std::vector<column_name_info>{}}
          : device_json_column_to_cudf_column(json_col.child_columns.begin()->second,
                                              d_input,
                                              options,
                                              prune_columns,
                                              child_schema_element,
                                              stream,
                                              mr);
      column_names.back().children      = names;
      auto [result_bitmask, null_count] = make_validity(json_col);
      auto ret_col                      = make_lists_column(num_rows,
                                       std::move(offsets_column),
                                       std::move(child_column),
                                       0,
                                       rmm::device_buffer{0, stream, mr},
                                       stream,
                                       mr);
      // The null_mask is set after creation of list column is to skip the purge_nonempty_nulls and
      // null validation applied in make_lists_column factory, which is not needed for json
      // parent column cannot be null when its children is non-empty in JSON
      if (null_count != 0) { ret_col->set_null_mask(std::move(result_bitmask), null_count); }
      return {std::move(ret_col), std::move(column_names)};
    }
    default: CUDF_FAIL("Unsupported column type"); break;
  }
}

table_with_metadata device_parse_nested_json(device_span<SymbolT const> d_input,
                                             cudf::io::json_reader_options const& options,
                                             rmm::cuda_stream_view stream,
                                             rmm::device_async_resource_ref mr)
{
  CUDF_FUNC_RANGE();

  auto gpu_tree = [&]() {
    // Parse the JSON and get the token stream
    const auto [tokens_gpu, token_indices_gpu] =
      get_token_stream(d_input, options, stream, cudf::get_current_device_resource_ref());
    // gpu tree generation
    // Note that to normalize whitespaces in nested columns coerced to be string, we need the column
    // to either be of mixed type or we need to request the column to be returned as string by
    // pruning it with the STRING dtype
    return get_tree_representation(
      tokens_gpu,
      token_indices_gpu,
      options.is_enabled_mixed_types_as_string() || options.is_enabled_prune_columns(),
      stream,
      cudf::get_current_device_resource_ref());
  }();  // IILE used to free memory of token data.
#ifdef NJP_DEBUG_PRINT
  auto h_input = cudf::detail::make_host_vector_async(d_input, stream);
  print_tree(h_input, gpu_tree, stream);
#endif

  bool const is_array_of_arrays = [&]() {
    std::array<node_t, 2> h_node_categories = {NC_ERR, NC_ERR};
    auto const size_to_copy                 = std::min(size_t{2}, gpu_tree.node_categories.size());
    CUDF_CUDA_TRY(hipMemcpyAsync(h_node_categories.data(),
                                  gpu_tree.node_categories.data(),
                                  sizeof(node_t) * size_to_copy,
                                  hipMemcpyDefault,
                                  stream.value()));
    stream.synchronize();
    if (options.is_enabled_lines()) return h_node_categories[0] == NC_LIST;
    return h_node_categories[0] == NC_LIST and h_node_categories[1] == NC_LIST;
  }();

  auto [gpu_col_id, gpu_row_offsets] =
    records_orient_tree_traversal(d_input,
                                  gpu_tree,
                                  is_array_of_arrays,
                                  options.is_enabled_lines(),
                                  stream,
                                  cudf::get_current_device_resource_ref());

  device_json_column root_column(stream, mr);
  root_column.type = json_col_t::ListColumn;
  root_column.child_offsets.resize(2, stream);
  thrust::fill(rmm::exec_policy(stream),
               root_column.child_offsets.begin(),
               root_column.child_offsets.end(),
               0);

  // Get internal JSON column
  make_device_json_column(d_input,
                          gpu_tree,
                          gpu_col_id,
                          gpu_row_offsets,
                          root_column,
                          is_array_of_arrays,
                          options,
                          stream,
                          mr);

  // data_root refers to the root column of the data represented by the given JSON string
  auto& data_root =
    options.is_enabled_lines() ? root_column : root_column.child_columns.begin()->second;

  // Zero row entries
  if (data_root.type == json_col_t::ListColumn && data_root.child_columns.empty()) {
    return table_with_metadata{std::make_unique<table>(std::vector<std::unique_ptr<column>>{})};
  }

  // Verify that we were in fact given a list of structs (or in JSON speech: an array of objects)
  auto constexpr single_child_col_count = 1;
  CUDF_EXPECTS(data_root.type == json_col_t::ListColumn and
                 data_root.child_columns.size() == single_child_col_count and
                 data_root.child_columns.begin()->second.type ==
                   (is_array_of_arrays ? json_col_t::ListColumn : json_col_t::StructColumn),
               "Input needs to be an array of arrays or an array of (nested) objects");

  // Slice off the root list column, which has only a single row that contains all the structs
  auto& root_struct_col = data_root.child_columns.begin()->second;

  // Initialize meta data to be populated while recursing through the tree of columns
  std::vector<std::unique_ptr<column>> out_columns;
  std::vector<column_name_info> out_column_names;
  auto parse_opt = parsing_options(options, stream);

  // Iterate over the struct's child columns and convert to cudf column
  size_type column_index = 0;
  for (auto const& col_name : root_struct_col.column_order) {
    auto& json_col = root_struct_col.child_columns.find(col_name)->second;

    std::optional<schema_element> child_schema_element = std::visit(
      cudf::detail::visitor_overload{
        [column_index](std::vector<data_type> const& user_dtypes) -> std::optional<schema_element> {
          return (static_cast<std::size_t>(column_index) < user_dtypes.size())
                   ? std::optional<schema_element>{{user_dtypes[column_index]}}
                   : std::optional<schema_element>{};
        },
        [col_name](
          std::map<std::string, data_type> const& user_dtypes) -> std::optional<schema_element> {
          return (user_dtypes.find(col_name) != std::end(user_dtypes))
                   ? std::optional<schema_element>{{user_dtypes.find(col_name)->second}}
                   : std::optional<schema_element>{};
        },
        [col_name](std::map<std::string, schema_element> const& user_dtypes)
          -> std::optional<schema_element> {
          return (user_dtypes.find(col_name) != std::end(user_dtypes))
                   ? user_dtypes.find(col_name)->second
                   : std::optional<schema_element>{};
        }},
      options.get_dtypes());
#ifdef NJP_DEBUG_PRINT
    auto debug_schema_print = [](auto ret) {
      std::cout << ", type id: "
                << (ret.has_value() ? std::to_string(static_cast<int>(ret->type.id())) : "n/a")
                << ", with " << (ret.has_value() ? ret->child_types.size() : 0) << " children"
                << "\n";
    };
    std::visit(
      cudf::detail::visitor_overload{[column_index](std::vector<data_type> const&) {
                                       std::cout << "Column by index: #" << column_index;
                                     },
                                     [col_name](std::map<std::string, data_type> const&) {
                                       std::cout << "Column by flat name: '" << col_name;
                                     },
                                     [col_name](std::map<std::string, schema_element> const&) {
                                       std::cout << "Column by nested name: #" << col_name;
                                     }},
      options.get_dtypes());
    debug_schema_print(child_schema_element);
#endif

    if (!options.is_enabled_prune_columns() or child_schema_element.has_value()) {
      // Get this JSON column's cudf column and schema info, (modifies json_col)
      auto [cudf_col, col_name_info] =
        device_json_column_to_cudf_column(json_col,
                                          d_input,
                                          parse_opt,
                                          options.is_enabled_prune_columns(),
                                          child_schema_element,
                                          stream,
                                          mr);
      // Insert this column's name into the schema
      out_column_names.emplace_back(col_name);
      // TODO: RangeIndex as DataFrame.columns names for array of arrays
      // if (is_array_of_arrays) {
      //   col_name_info.back().name = "";
      // }

      out_column_names.back().children = std::move(col_name_info);
      out_columns.emplace_back(std::move(cudf_col));

      column_index++;
    }
  }

  return table_with_metadata{std::make_unique<table>(std::move(out_columns)), {out_column_names}};
}

}  // namespace cudf::io::json::detail
