#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2022-2024, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "io/utilities/parsing_utils.cuh"
#include "io/utilities/string_parsing.hpp"
#include "json_utils.hpp"
#include "nested_json.hpp"

#include <cudf/column/column_factories.hpp>
#include <cudf/detail/null_mask.hpp>
#include <cudf/detail/nvtx/ranges.hpp>
#include <cudf/detail/utilities/vector_factories.hpp>
#include <cudf/detail/utilities/visitor_overload.hpp>
#include <cudf/strings/strings_column_view.hpp>
#include <cudf/types.hpp>
#include <cudf/utilities/error.hpp>
#include <cudf/utilities/span.hpp>

#include <rmm/cuda_stream_view.hpp>
#include <rmm/device_uvector.hpp>
#include <rmm/exec_policy.hpp>
#include <rmm/resource_ref.hpp>

#include <cuda/atomic>
#include <cuda/functional>
#include <thrust/count.h>
#include <thrust/for_each.h>
#include <thrust/functional.h>
#include <thrust/gather.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/iterator/discard_iterator.h>
#include <thrust/iterator/permutation_iterator.h>
#include <thrust/iterator/zip_iterator.h>
#include <thrust/reduce.h>
#include <thrust/scan.h>
#include <thrust/sort.h>
#include <thrust/transform.h>
#include <thrust/unique.h>

#include <algorithm>
#include <cstdint>

namespace cudf::io::json::detail {

// DEBUG prints
auto to_cat = [](auto v) -> std::string {
  switch (v) {
    case NC_STRUCT: return " S";
    case NC_LIST: return " L";
    case NC_STR: return " \"";
    case NC_VAL: return " V";
    case NC_FN: return " F";
    case NC_ERR: return "ER";
    default: return "UN";
  };
};
auto to_int    = [](auto v) { return std::to_string(static_cast<int>(v)); };
auto print_vec = [](auto const& cpu, auto const name, auto converter) {
  for (auto const& v : cpu)
    printf("%3s,", converter(v).c_str());
  std::cout << name << std::endl;
};

void print_tree(host_span<SymbolT const> input,
                tree_meta_t const& d_gpu_tree,
                rmm::cuda_stream_view stream)
{
  print_vec(cudf::detail::make_std_vector_sync(d_gpu_tree.node_categories, stream),
            "node_categories",
            to_cat);
  print_vec(cudf::detail::make_std_vector_sync(d_gpu_tree.parent_node_ids, stream),
            "parent_node_ids",
            to_int);
  print_vec(
    cudf::detail::make_std_vector_sync(d_gpu_tree.node_levels, stream), "node_levels", to_int);
  auto node_range_begin = cudf::detail::make_std_vector_sync(d_gpu_tree.node_range_begin, stream);
  auto node_range_end   = cudf::detail::make_std_vector_sync(d_gpu_tree.node_range_end, stream);
  print_vec(node_range_begin, "node_range_begin", to_int);
  print_vec(node_range_end, "node_range_end", to_int);
  for (int i = 0; i < int(node_range_begin.size()); i++) {
    printf("%3s ",
           std::string(input.data() + node_range_begin[i], node_range_end[i] - node_range_begin[i])
             .c_str());
  }
  printf(" (JSON)\n");
}

std::pair<std::unique_ptr<column>, std::vector<column_name_info>> device_json_column_to_cudf_column(
  device_json_column& json_col,
  device_span<SymbolT const> d_input,
  cudf::io::parse_options const& options,
  bool prune_columns,
  std::optional<schema_element> schema,
  rmm::cuda_stream_view stream,
  rmm::device_async_resource_ref mr)
{
  CUDF_FUNC_RANGE();
  auto validity_size_check = [](device_json_column& json_col) {
    CUDF_EXPECTS(json_col.validity.size() >= bitmask_allocation_size_bytes(json_col.num_rows),
                 "valid_count is too small");
  };
  auto make_validity = [stream, validity_size_check](
                         device_json_column& json_col) -> std::pair<rmm::device_buffer, size_type> {
    validity_size_check(json_col);
    auto null_count = cudf::detail::null_count(
      static_cast<bitmask_type*>(json_col.validity.data()), 0, json_col.num_rows, stream);
    // full null_mask is always required for parse_data
    return {std::move(json_col.validity), null_count};
    // Note: json_col modified here, moves this memory
  };

  auto get_child_schema = [schema](auto child_name) -> std::optional<schema_element> {
    if (schema.has_value()) {
      auto const result = schema.value().child_types.find(child_name);
      if (result != std::end(schema.value().child_types)) { return result->second; }
    }
    return {};
  };

  switch (json_col.type) {
    case json_col_t::StringColumn: {
      // move string_offsets to GPU and transform to string column
      auto const col_size      = json_col.string_offsets.size();
      using char_length_pair_t = thrust::pair<char const*, size_type>;
      CUDF_EXPECTS(json_col.string_offsets.size() == json_col.string_lengths.size(),
                   "string offset, string length mismatch");
      rmm::device_uvector<char_length_pair_t> d_string_data(col_size, stream);
      // TODO how about directly storing pair<char*, size_t> in json_column?
      auto offset_length_it =
        thrust::make_zip_iterator(json_col.string_offsets.begin(), json_col.string_lengths.begin());

      data_type target_type{};

      if (schema.has_value()) {
#ifdef NJP_DEBUG_PRINT
        std::cout << "-> explicit type: "
                  << (schema.has_value() ? std::to_string(static_cast<int>(schema->type.id()))
                                         : "n/a");
#endif
        target_type = schema.value().type;
      } else if (json_col.forced_as_string_column) {
        target_type = data_type{type_id::STRING};
      }
      // Infer column type, if we don't have an explicit type for it
      else {
        target_type = cudf::io::detail::infer_data_type(
          options.json_view(), d_input, offset_length_it, col_size, stream);
      }

      auto [result_bitmask, null_count] = make_validity(json_col);
      // Convert strings to the inferred data type
      auto col = parse_data(d_input.data(),
                            offset_length_it,
                            col_size,
                            target_type,
                            std::move(result_bitmask),
                            null_count,
                            options.view(),
                            stream,
                            mr);

      // Reset nullable if we do not have nulls
      // This is to match the existing JSON reader's behaviour:
      // - Non-string columns will always be returned as nullable
      // - String columns will be returned as nullable, iff there's at least one null entry
      if (col->null_count() == 0) { col->set_null_mask(rmm::device_buffer{0, stream, mr}, 0); }

      // For string columns return ["offsets", "char"] schema
      if (target_type.id() == type_id::STRING) {
        return {std::move(col), std::vector<column_name_info>{{"offsets"}, {"chars"}}};
      }
      // Non-string leaf-columns (e.g., numeric) do not have child columns in the schema
      return {std::move(col), std::vector<column_name_info>{}};
    }
    case json_col_t::StructColumn: {
      std::vector<std::unique_ptr<column>> child_columns;
      std::vector<column_name_info> column_names{};
      size_type num_rows{json_col.num_rows};
      // Create children columns
      for (auto const& col_name : json_col.column_order) {
        auto const& col = json_col.child_columns.find(col_name);
        column_names.emplace_back(col->first);
        auto& child_col           = col->second;
        auto child_schema_element = get_child_schema(col_name);
        if (!prune_columns or child_schema_element.has_value()) {
          auto [child_column, names] = device_json_column_to_cudf_column(
            child_col, d_input, options, prune_columns, child_schema_element, stream, mr);
          CUDF_EXPECTS(num_rows == child_column->size(),
                       "All children columns must have the same size");
          child_columns.push_back(std::move(child_column));
          column_names.back().children = names;
        }
      }
      auto [result_bitmask, null_count] = make_validity(json_col);
      // The null_mask is set after creation of struct column is to skip the superimpose_nulls and
      // null validation applied in make_structs_column factory, which is not needed for json
      auto ret_col = make_structs_column(num_rows, std::move(child_columns), 0, {}, stream, mr);
      if (null_count != 0) { ret_col->set_null_mask(std::move(result_bitmask), null_count); }
      return {std::move(ret_col), column_names};
    }
    case json_col_t::ListColumn: {
      size_type num_rows = json_col.child_offsets.size() - 1;
      std::vector<column_name_info> column_names{};
      column_names.emplace_back("offsets");
      column_names.emplace_back(
        json_col.child_columns.empty() ? list_child_name : json_col.child_columns.begin()->first);

      // Note: json_col modified here, reuse the memory
      auto offsets_column = std::make_unique<column>(data_type{type_id::INT32},
                                                     num_rows + 1,
                                                     json_col.child_offsets.release(),
                                                     rmm::device_buffer{},
                                                     0);
      // Create children column
      auto child_schema_element = json_col.child_columns.empty()
                                    ? std::optional<schema_element>{}
                                    : get_child_schema(json_col.child_columns.begin()->first);
      auto [child_column, names] =
        json_col.child_columns.empty() or (prune_columns and !child_schema_element.has_value())
          ? std::pair<std::unique_ptr<column>,
                      // EMPTY type could not used because gather throws exception on EMPTY type.
                      std::vector<column_name_info>>{std::make_unique<column>(
                                                       data_type{type_id::INT8},
                                                       0,
                                                       rmm::device_buffer{},
                                                       rmm::device_buffer{},
                                                       0),
                                                     std::vector<column_name_info>{}}
          : device_json_column_to_cudf_column(json_col.child_columns.begin()->second,
                                              d_input,
                                              options,
                                              prune_columns,
                                              child_schema_element,
                                              stream,
                                              mr);
      column_names.back().children      = names;
      auto [result_bitmask, null_count] = make_validity(json_col);
      auto ret_col                      = make_lists_column(num_rows,
                                       std::move(offsets_column),
                                       std::move(child_column),
                                       0,
                                       rmm::device_buffer{0, stream, mr},
                                       stream,
                                       mr);
      // The null_mask is set after creation of list column is to skip the purge_nonempty_nulls and
      // null validation applied in make_lists_column factory, which is not needed for json
      // parent column cannot be null when its children is non-empty in JSON
      if (null_count != 0) { ret_col->set_null_mask(std::move(result_bitmask), null_count); }
      return {std::move(ret_col), std::move(column_names)};
    }
    default: CUDF_FAIL("Unsupported column type"); break;
  }
}

table_with_metadata device_parse_nested_json(device_span<SymbolT const> d_input,
                                             cudf::io::json_reader_options const& options,
                                             rmm::cuda_stream_view stream,
                                             rmm::device_async_resource_ref mr)
{
  CUDF_FUNC_RANGE();

  auto gpu_tree = [&]() {
    // Parse the JSON and get the token stream
    const auto [tokens_gpu, token_indices_gpu] =
      get_token_stream(d_input, options, stream, rmm::mr::get_current_device_resource());
    // gpu tree generation
    return get_tree_representation(tokens_gpu,
                                   token_indices_gpu,
                                   options.is_enabled_mixed_types_as_string(),
                                   stream,
                                   rmm::mr::get_current_device_resource());
  }();  // IILE used to free memory of token data.
#ifdef NJP_DEBUG_PRINT
  auto h_input = cudf::detail::make_host_vector_async(d_input, stream);
  print_tree(h_input, gpu_tree, stream);
#endif

  bool const is_array_of_arrays = [&]() {
    std::array<node_t, 2> h_node_categories = {NC_ERR, NC_ERR};
    auto const size_to_copy                 = std::min(size_t{2}, gpu_tree.node_categories.size());
    CUDF_CUDA_TRY(hipMemcpyAsync(h_node_categories.data(),
                                  gpu_tree.node_categories.data(),
                                  sizeof(node_t) * size_to_copy,
                                  hipMemcpyDefault,
                                  stream.value()));
    stream.synchronize();
    if (options.is_enabled_lines()) return h_node_categories[0] == NC_LIST;
    return h_node_categories[0] == NC_LIST and h_node_categories[1] == NC_LIST;
  }();

  auto [gpu_col_id, gpu_row_offsets] =
    records_orient_tree_traversal(d_input,
                                  gpu_tree,
                                  is_array_of_arrays,
                                  options.is_enabled_lines(),
                                  stream,
                                  rmm::mr::get_current_device_resource());

  device_json_column root_column(stream, mr);
  root_column.type = json_col_t::ListColumn;
  root_column.child_offsets.resize(2, stream);
  thrust::fill(rmm::exec_policy(stream),
               root_column.child_offsets.begin(),
               root_column.child_offsets.end(),
               0);

  // Get internal JSON column
  make_device_json_column(d_input,
                          gpu_tree,
                          gpu_col_id,
                          gpu_row_offsets,
                          root_column,
                          is_array_of_arrays,
                          options,
                          stream,
                          mr);

  // data_root refers to the root column of the data represented by the given JSON string
  auto& data_root =
    options.is_enabled_lines() ? root_column : root_column.child_columns.begin()->second;

  // Zero row entries
  if (data_root.type == json_col_t::ListColumn && data_root.child_columns.empty()) {
    return table_with_metadata{std::make_unique<table>(std::vector<std::unique_ptr<column>>{})};
  }

  // Verify that we were in fact given a list of structs (or in JSON speech: an array of objects)
  auto constexpr single_child_col_count = 1;
  CUDF_EXPECTS(data_root.type == json_col_t::ListColumn and
                 data_root.child_columns.size() == single_child_col_count and
                 data_root.child_columns.begin()->second.type ==
                   (is_array_of_arrays ? json_col_t::ListColumn : json_col_t::StructColumn),
               "Input needs to be an array of arrays or an array of (nested) objects");

  // Slice off the root list column, which has only a single row that contains all the structs
  auto& root_struct_col = data_root.child_columns.begin()->second;

  // Initialize meta data to be populated while recursing through the tree of columns
  std::vector<std::unique_ptr<column>> out_columns;
  std::vector<column_name_info> out_column_names;
  auto parse_opt = parsing_options(options, stream);

  // Iterate over the struct's child columns and convert to cudf column
  size_type column_index = 0;
  for (auto const& col_name : root_struct_col.column_order) {
    auto& json_col = root_struct_col.child_columns.find(col_name)->second;

    std::optional<schema_element> child_schema_element = std::visit(
      cudf::detail::visitor_overload{
        [column_index](std::vector<data_type> const& user_dtypes) -> std::optional<schema_element> {
          return (static_cast<std::size_t>(column_index) < user_dtypes.size())
                   ? std::optional<schema_element>{{user_dtypes[column_index]}}
                   : std::optional<schema_element>{};
        },
        [col_name](
          std::map<std::string, data_type> const& user_dtypes) -> std::optional<schema_element> {
          return (user_dtypes.find(col_name) != std::end(user_dtypes))
                   ? std::optional<schema_element>{{user_dtypes.find(col_name)->second}}
                   : std::optional<schema_element>{};
        },
        [col_name](std::map<std::string, schema_element> const& user_dtypes)
          -> std::optional<schema_element> {
          return (user_dtypes.find(col_name) != std::end(user_dtypes))
                   ? user_dtypes.find(col_name)->second
                   : std::optional<schema_element>{};
        }},
      options.get_dtypes());
#ifdef NJP_DEBUG_PRINT
    auto debug_schema_print = [](auto ret) {
      std::cout << ", type id: "
                << (ret.has_value() ? std::to_string(static_cast<int>(ret->type.id())) : "n/a")
                << ", with " << (ret.has_value() ? ret->child_types.size() : 0) << " children"
                << "\n";
    };
    std::visit(
      cudf::detail::visitor_overload{[column_index](std::vector<data_type> const&) {
                                       std::cout << "Column by index: #" << column_index;
                                     },
                                     [col_name](std::map<std::string, data_type> const&) {
                                       std::cout << "Column by flat name: '" << col_name;
                                     },
                                     [col_name](std::map<std::string, schema_element> const&) {
                                       std::cout << "Column by nested name: #" << col_name;
                                     }},
      options.get_dtypes());
    debug_schema_print(child_schema_element);
#endif

    if (!options.is_enabled_prune_columns() or child_schema_element.has_value()) {
      // Get this JSON column's cudf column and schema info, (modifies json_col)
      auto [cudf_col, col_name_info] =
        device_json_column_to_cudf_column(json_col,
                                          d_input,
                                          parse_opt,
                                          options.is_enabled_prune_columns(),
                                          child_schema_element,
                                          stream,
                                          mr);
      // Insert this column's name into the schema
      out_column_names.emplace_back(col_name);
      // TODO: RangeIndex as DataFrame.columns names for array of arrays
      // if (is_array_of_arrays) {
      //   col_name_info.back().name = "";
      // }

      out_column_names.back().children = std::move(col_name_info);
      out_columns.emplace_back(std::move(cudf_col));

      column_index++;
    }
  }

  return table_with_metadata{std::make_unique<table>(std::move(out_columns)), {out_column_names}};
}

}  // namespace cudf::io::json::detail
