#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2022, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "nested_json.hpp"
#include <io/utilities/parsing_utils.cuh>
#include <io/utilities/type_inference.cuh>

#include <cudf/column/column_factories.hpp>
#include <cudf/detail/null_mask.hpp>
#include <cudf/detail/nvtx/ranges.hpp>
#include <cudf/detail/utilities/vector_factories.hpp>
#include <cudf/detail/utilities/visitor_overload.hpp>
#include <cudf/io/detail/data_casting.cuh>
#include <cudf/strings/strings_column_view.hpp>
#include <cudf/types.hpp>
#include <cudf/utilities/error.hpp>
#include <cudf/utilities/span.hpp>

#include <rmm/cuda_stream_view.hpp>
#include <rmm/device_uvector.hpp>
#include <rmm/exec_policy.hpp>

#include <thrust/count.h>
#include <thrust/for_each.h>
#include <thrust/functional.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/iterator/discard_iterator.h>
#include <thrust/iterator/permutation_iterator.h>
#include <thrust/iterator/zip_iterator.h>
#include <thrust/reduce.h>
#include <thrust/scan.h>
#include <thrust/sort.h>
#include <thrust/transform.h>
#include <thrust/unique.h>

#include <algorithm>
#include <cstdint>

namespace cudf::io::json {
namespace detail {

// DEBUG prints
auto to_cat = [](auto v) -> std::string {
  switch (v) {
    case NC_STRUCT: return " S";
    case NC_LIST: return " L";
    case NC_STR: return " \"";
    case NC_VAL: return " V";
    case NC_FN: return " F";
    case NC_ERR: return "ER";
    default: return "UN";
  };
};
auto to_int    = [](auto v) { return std::to_string(static_cast<int>(v)); };
auto print_vec = [](auto const& cpu, auto const name, auto converter) {
  for (auto const& v : cpu)
    printf("%3s,", converter(v).c_str());
  std::cout << name << std::endl;
};

void print_tree(host_span<SymbolT const> input,
                tree_meta_t const& d_gpu_tree,
                rmm::cuda_stream_view stream)
{
  print_vec(cudf::detail::make_std_vector_async(d_gpu_tree.node_categories, stream),
            "node_categories",
            to_cat);
  print_vec(cudf::detail::make_std_vector_async(d_gpu_tree.parent_node_ids, stream),
            "parent_node_ids",
            to_int);
  print_vec(
    cudf::detail::make_std_vector_async(d_gpu_tree.node_levels, stream), "node_levels", to_int);
  auto node_range_begin = cudf::detail::make_std_vector_async(d_gpu_tree.node_range_begin, stream);
  auto node_range_end   = cudf::detail::make_std_vector_async(d_gpu_tree.node_range_end, stream);
  print_vec(node_range_begin, "node_range_begin", to_int);
  print_vec(node_range_end, "node_range_end", to_int);
  for (int i = 0; i < int(node_range_begin.size()); i++) {
    printf("%3s ",
           std::string(input.data() + node_range_begin[i], node_range_end[i] - node_range_begin[i])
             .c_str());
  }
  printf(" (JSON)\n");
}

/**
 * @brief Reduces node tree representation to column tree representation.
 *
 * @param tree Node tree representation of JSON string
 * @param col_ids Column ids of nodes
 * @param row_offsets Row offsets of nodes
 * @param stream CUDA stream used for device memory operations and kernel launches
 * @return A tuple of column tree representation of JSON string, column ids of columns, and
 * max row offsets of columns
 */
std::tuple<tree_meta_t, rmm::device_uvector<NodeIndexT>, rmm::device_uvector<size_type>>
reduce_to_column_tree(tree_meta_t& tree,
                      device_span<NodeIndexT> col_ids,
                      device_span<size_type> row_offsets,
                      rmm::cuda_stream_view stream)
{
  CUDF_FUNC_RANGE();
  //   1. sort_by_key {col_id}, {row_offset} stable
  rmm::device_uvector<NodeIndexT> node_ids(row_offsets.size(), stream);
  thrust::sequence(rmm::exec_policy(stream), node_ids.begin(), node_ids.end());
  thrust::stable_sort_by_key(rmm::exec_policy(stream),
                             col_ids.begin(),
                             col_ids.end(),
                             thrust::make_zip_iterator(node_ids.begin(), row_offsets.begin()));
  auto num_columns = thrust::unique_count(rmm::exec_policy(stream), col_ids.begin(), col_ids.end());

  // 2. reduce_by_key {col_id}, {row_offset}, max.
  rmm::device_uvector<NodeIndexT> unique_col_ids(num_columns, stream);
  rmm::device_uvector<size_type> max_row_offsets(num_columns, stream);
  thrust::reduce_by_key(rmm::exec_policy(stream),
                        col_ids.begin(),
                        col_ids.end(),
                        row_offsets.begin(),
                        unique_col_ids.begin(),
                        max_row_offsets.begin(),
                        thrust::equal_to<size_type>(),
                        thrust::maximum<size_type>());

  // 3. reduce_by_key {col_id}, {node_categories} - custom opp (*+v=*, v+v=v, *+#=E)
  rmm::device_uvector<NodeT> column_categories(num_columns, stream);
  thrust::reduce_by_key(
    rmm::exec_policy(stream),
    col_ids.begin(),
    col_ids.end(),
    thrust::make_permutation_iterator(tree.node_categories.begin(), node_ids.begin()),
    unique_col_ids.begin(),
    column_categories.begin(),
    thrust::equal_to<size_type>(),
    [] __device__(NodeT type_a, NodeT type_b) -> NodeT {
      auto is_a_leaf = (type_a == NC_VAL || type_a == NC_STR);
      auto is_b_leaf = (type_b == NC_VAL || type_b == NC_STR);
      // (v+v=v, *+*=*,  *+v=*, *+#=E, NESTED+VAL=NESTED)
      // *+*=*, v+v=v
      if (type_a == type_b) {
        return type_a;
      } else if (is_a_leaf) {
        // *+v=*, N+V=N
        // STRUCT/LIST + STR/VAL = STRUCT/LIST, STR/VAL + FN = ERR, STR/VAL + STR = STR
        return type_b == NC_FN ? NC_ERR : (is_b_leaf ? NC_STR : type_b);
      } else if (is_b_leaf) {
        return type_a == NC_FN ? NC_ERR : (is_a_leaf ? NC_STR : type_a);
      }
      // *+#=E
      return NC_ERR;
    });

  // 4. unique_copy parent_node_ids, ranges
  rmm::device_uvector<TreeDepthT> column_levels(0, stream);  // not required
  rmm::device_uvector<NodeIndexT> parent_col_ids(num_columns, stream);
  rmm::device_uvector<SymbolOffsetT> col_range_begin(num_columns, stream);  // Field names
  rmm::device_uvector<SymbolOffsetT> col_range_end(num_columns, stream);
  rmm::device_uvector<size_type> unique_node_ids(num_columns, stream);
  thrust::unique_by_key_copy(rmm::exec_policy(stream),
                             col_ids.begin(),
                             col_ids.end(),
                             node_ids.begin(),
                             thrust::make_discard_iterator(),
                             unique_node_ids.begin());
  thrust::copy_n(
    rmm::exec_policy(stream),
    thrust::make_zip_iterator(
      thrust::make_permutation_iterator(tree.parent_node_ids.begin(), unique_node_ids.begin()),
      thrust::make_permutation_iterator(tree.node_range_begin.begin(), unique_node_ids.begin()),
      thrust::make_permutation_iterator(tree.node_range_end.begin(), unique_node_ids.begin())),
    unique_node_ids.size(),
    thrust::make_zip_iterator(
      parent_col_ids.begin(), col_range_begin.begin(), col_range_end.begin()));

  // Restore the order
  {
    // use scatter to restore the order
    rmm::device_uvector<NodeIndexT> temp_col_ids(col_ids.size(), stream);
    rmm::device_uvector<size_type> temp_row_offsets(row_offsets.size(), stream);
    thrust::scatter(rmm::exec_policy(stream),
                    thrust::make_zip_iterator(col_ids.begin(), row_offsets.begin()),
                    thrust::make_zip_iterator(col_ids.end(), row_offsets.end()),
                    node_ids.begin(),
                    thrust::make_zip_iterator(temp_col_ids.begin(), temp_row_offsets.begin()));
    thrust::copy(rmm::exec_policy(stream),
                 thrust::make_zip_iterator(temp_col_ids.begin(), temp_row_offsets.begin()),
                 thrust::make_zip_iterator(temp_col_ids.end(), temp_row_offsets.end()),
                 thrust::make_zip_iterator(col_ids.begin(), row_offsets.begin()));
  }

  // convert parent_node_ids to parent_col_ids
  thrust::transform(rmm::exec_policy(stream),
                    parent_col_ids.begin(),
                    parent_col_ids.end(),
                    parent_col_ids.begin(),
                    [col_ids = col_ids.begin()] __device__(auto parent_node_id) -> size_type {
                      return parent_node_id == parent_node_sentinel ? parent_node_sentinel
                                                                    : col_ids[parent_node_id];
                    });

  // copy lists' max_row_offsets to children.
  // all structs should have same size.
  thrust::transform_if(
    rmm::exec_policy(stream),
    unique_col_ids.begin(),
    unique_col_ids.end(),
    max_row_offsets.begin(),
    [column_categories = column_categories.begin(),
     parent_col_ids    = parent_col_ids.begin(),
     max_row_offsets   = max_row_offsets.begin()] __device__(size_type col_id) {
      auto parent_col_id = parent_col_ids[col_id];
      while (parent_col_id != parent_node_sentinel and
             column_categories[parent_col_id] != node_t::NC_LIST) {
        col_id        = parent_col_id;
        parent_col_id = parent_col_ids[parent_col_id];
      }
      return max_row_offsets[col_id];
    },
    [column_categories = column_categories.begin(),
     parent_col_ids    = parent_col_ids.begin()] __device__(size_type col_id) {
      auto parent_col_id = parent_col_ids[col_id];
      return parent_col_id != parent_node_sentinel and
             (column_categories[parent_col_id] != node_t::NC_LIST);
      // Parent is not a list, or sentinel/root
    });

  return std::tuple{tree_meta_t{std::move(column_categories),
                                std::move(parent_col_ids),
                                std::move(column_levels),
                                std::move(col_range_begin),
                                std::move(col_range_end)},
                    std::move(unique_col_ids),
                    std::move(max_row_offsets)};
}

/**
 * @brief Copies strings specified by pair of begin, end offsets to host vector of strings.
 *
 * @param input String device buffer
 * @param node_range_begin Begin offset of the strings
 * @param node_range_end End offset of the strings
 * @param stream CUDA stream
 * @return Vector of strings
 */
std::vector<std::string> copy_strings_to_host(device_span<SymbolT const> input,
                                              device_span<SymbolOffsetT const> node_range_begin,
                                              device_span<SymbolOffsetT const> node_range_end,
                                              rmm::cuda_stream_view stream)
{
  CUDF_FUNC_RANGE();
  auto const num_strings = node_range_begin.size();
  rmm::device_uvector<thrust::pair<const char*, size_type>> string_views(num_strings, stream);
  auto d_offset_pairs = thrust::make_zip_iterator(node_range_begin.begin(), node_range_end.begin());
  thrust::transform(rmm::exec_policy(stream),
                    d_offset_pairs,
                    d_offset_pairs + num_strings,
                    string_views.begin(),
                    [data = input.data()] __device__(auto const& offsets) {
                      // Note: first character for non-field columns
                      return thrust::make_pair(
                        data + thrust::get<0>(offsets),
                        static_cast<size_type>(thrust::get<1>(offsets) - thrust::get<0>(offsets)));
                    });
  auto d_column_names = cudf::make_strings_column(string_views, stream);
  auto to_host        = [](auto const& col) {
    if (col.is_empty()) return std::vector<std::string>{};
    auto const scv     = cudf::strings_column_view(col);
    auto const h_chars = cudf::detail::make_std_vector_sync<char>(
      cudf::device_span<char const>(scv.chars().data<char>(), scv.chars().size()),
      cudf::get_default_stream());
    auto const h_offsets = cudf::detail::make_std_vector_sync(
      cudf::device_span<cudf::offset_type const>(
        scv.offsets().data<cudf::offset_type>() + scv.offset(), scv.size() + 1),
      cudf::get_default_stream());

    // build std::string vector from chars and offsets
    std::vector<std::string> host_data;
    host_data.reserve(col.size());
    std::transform(
      std::begin(h_offsets),
      std::end(h_offsets) - 1,
      std::begin(h_offsets) + 1,
      std::back_inserter(host_data),
      [&](auto start, auto end) { return std::string(h_chars.data() + start, end - start); });
    return host_data;
  };
  return to_host(d_column_names->view());
}

/**
 * @brief Holds member data pointers of `d_json_column`
 *
 */
struct json_column_data {
  using row_offset_t = json_column::row_offset_t;
  row_offset_t* string_offsets;
  row_offset_t* string_lengths;
  row_offset_t* child_offsets;
  bitmask_type* validity;
};

/**
 * @brief Constructs `d_json_column` from node tree representation
 * Newly constructed columns are insert into `root`'s children.
 * `root` must be a list type.
 *
 * @param input Input JSON string device data
 * @param tree Node tree representation of the JSON string
 * @param col_ids Column ids of the nodes in the tree
 * @param row_offsets Row offsets of the nodes in the tree
 * @param root Root node of the `d_json_column` tree
 * @param stream CUDA stream used for device memory operations and kernel launches
 * @param mr Device memory resource used to allocate the device memory
 * of child_offets and validity members of `d_json_column`
 */
void make_device_json_column(device_span<SymbolT const> input,
                             tree_meta_t& tree,
                             device_span<NodeIndexT> col_ids,
                             device_span<size_type> row_offsets,
                             device_json_column& root,
                             rmm::cuda_stream_view stream,
                             rmm::mr::device_memory_resource* mr)
{
  CUDF_FUNC_RANGE();
  // 1. gather column information.
  auto [d_column_tree, d_unique_col_ids, d_max_row_offsets] =
    reduce_to_column_tree(tree, col_ids, row_offsets, stream);
  auto num_columns    = d_unique_col_ids.size();
  auto unique_col_ids = cudf::detail::make_std_vector_async(d_unique_col_ids, stream);
  auto column_categories =
    cudf::detail::make_std_vector_async(d_column_tree.node_categories, stream);
  auto column_parent_ids =
    cudf::detail::make_std_vector_async(d_column_tree.parent_node_ids, stream);
  auto column_range_beg =
    cudf::detail::make_std_vector_async(d_column_tree.node_range_begin, stream);
  auto max_row_offsets = cudf::detail::make_std_vector_async(d_max_row_offsets, stream);
  std::vector<std::string> column_names = copy_strings_to_host(
    input, d_column_tree.node_range_begin, d_column_tree.node_range_end, stream);

  auto to_json_col_type = [](auto category) {
    switch (category) {
      case NC_STRUCT: return json_col_t::StructColumn;
      case NC_LIST: return json_col_t::ListColumn;
      case NC_STR:
      case NC_VAL: return json_col_t::StringColumn;
      default: return json_col_t::Unknown;
    }
  };
  auto init_to_zero = [stream](auto& v) {
    thrust::uninitialized_fill(rmm::exec_policy(stream), v.begin(), v.end(), 0);
  };

  auto initialize_json_columns = [&](auto i, auto& col) {
    if (column_categories[i] == NC_ERR || column_categories[i] == NC_FN) {
      return;
    } else if (column_categories[i] == NC_VAL || column_categories[i] == NC_STR) {
      col.string_offsets.resize(max_row_offsets[i] + 1, stream);
      col.string_lengths.resize(max_row_offsets[i] + 1, stream);
      init_to_zero(col.string_offsets);
      init_to_zero(col.string_lengths);
    } else if (column_categories[i] == NC_LIST) {
      col.child_offsets.resize(max_row_offsets[i] + 2, stream);
      init_to_zero(col.child_offsets);
    }
    col.num_rows = max_row_offsets[i] + 1;
    col.validity.resize(bitmask_allocation_size_bytes(max_row_offsets[i] + 1), stream);
    init_to_zero(col.validity);
    col.type = to_json_col_type(column_categories[i]);
  };

  // 2. generate nested columns tree and its device_memory
  // reorder unique_col_ids w.r.t. column_range_begin for order of column to be in field order.
  auto h_range_col_id_it =
    thrust::make_zip_iterator(column_range_beg.begin(), unique_col_ids.begin());
  std::sort(h_range_col_id_it, h_range_col_id_it + num_columns, [](auto const& a, auto const& b) {
    return thrust::get<0>(a) < thrust::get<0>(b);
  });

  // use hash map because we may skip field name's col_ids
  std::unordered_map<NodeIndexT, std::reference_wrapper<device_json_column>> columns;
  // map{parent_col_id, child_col_name}> = child_col_id, used for null value column tracking
  std::map<std::pair<NodeIndexT, std::string>, NodeIndexT> mapped_columns;
  // find column_ids which are values, but should be ignored in validity
  std::vector<uint8_t> ignore_vals(num_columns, 0);
  columns.try_emplace(parent_node_sentinel, std::ref(root));

  for (auto const this_col_id : unique_col_ids) {
    if (column_categories[this_col_id] == NC_ERR || column_categories[this_col_id] == NC_FN) {
      continue;
    }
    // Struct, List, String, Value
    std::string name   = "";
    auto parent_col_id = column_parent_ids[this_col_id];
    if (parent_col_id == parent_node_sentinel || column_categories[parent_col_id] == NC_LIST) {
      name = list_child_name;
    } else if (column_categories[parent_col_id] == NC_FN) {
      auto field_name_col_id = parent_col_id;
      parent_col_id          = column_parent_ids[parent_col_id];
      name                   = column_names[field_name_col_id];
    } else {
      CUDF_FAIL("Unexpected parent column category");
    }
    // If the child is already found,
    // replace if this column is a nested column and the existing was a value column
    // ignore this column if this column is a value column and the existing was a nested column
    auto it = columns.find(parent_col_id);
    CUDF_EXPECTS(it != columns.end(), "Parent column not found");
    auto& parent_col = it->second.get();
    bool replaced    = false;
    if (mapped_columns.count({parent_col_id, name}) > 0) {
      if (column_categories[this_col_id] == NC_VAL || column_categories[this_col_id] == NC_STR) {
        ignore_vals[this_col_id] = 1;
        continue;
      }
      auto old_col_id = mapped_columns[{parent_col_id, name}];
      if (column_categories[old_col_id] == NC_VAL || column_categories[old_col_id] == NC_STR) {
        // remap
        ignore_vals[old_col_id] = 1;
        mapped_columns.erase({parent_col_id, name});
        columns.erase(old_col_id);
        parent_col.child_columns.erase(name);
        replaced = true;  // to skip duplicate name in column_order
      } else {
        // If this is a nested column but we're trying to insert either (a) a list node into a
        // struct column or (b) a struct node into a list column, we fail
        CUDF_EXPECTS(not((column_categories[old_col_id] == NC_LIST and
                          column_categories[this_col_id] == NC_STRUCT) or
                         (column_categories[old_col_id] == NC_STRUCT and
                          column_categories[this_col_id] == NC_LIST)),
                     "A mix of lists and structs within the same column is not supported");
      }
    }
    CUDF_EXPECTS(parent_col.child_columns.count(name) == 0, "duplicate column name");
    // move into parent
    device_json_column col(stream, mr);
    initialize_json_columns(this_col_id, col);
    auto inserted = parent_col.child_columns.try_emplace(name, std::move(col)).second;
    CUDF_EXPECTS(inserted, "child column insertion failed, duplicate column name in the parent");
    if (not replaced) parent_col.column_order.push_back(name);
    columns.try_emplace(this_col_id, std::ref(parent_col.child_columns.at(name)));
    mapped_columns.try_emplace(std::make_pair(parent_col_id, name), this_col_id);
  }
  // restore unique_col_ids order
  std::sort(h_range_col_id_it, h_range_col_id_it + num_columns, [](auto const& a, auto const& b) {
    return thrust::get<1>(a) < thrust::get<1>(b);
  });
  // move columns data to device.
  std::vector<json_column_data> columns_data(num_columns);
  for (auto& [col_id, col_ref] : columns) {
    if (col_id == parent_node_sentinel) continue;
    auto& col            = col_ref.get();
    columns_data[col_id] = json_column_data{col.string_offsets.data(),
                                            col.string_lengths.data(),
                                            col.child_offsets.data(),
                                            col.validity.data()};
  }

  // 3. scatter string offsets to respective columns, set validity bits
  auto d_ignore_vals  = cudf::detail::make_device_uvector_async(ignore_vals, stream);
  auto d_columns_data = cudf::detail::make_device_uvector_async(columns_data, stream);
  thrust::for_each_n(
    rmm::exec_policy(stream),
    thrust::counting_iterator<size_type>(0),
    col_ids.size(),
    [node_categories = tree.node_categories.begin(),
     col_ids         = col_ids.begin(),
     row_offsets     = row_offsets.begin(),
     range_begin     = tree.node_range_begin.begin(),
     range_end       = tree.node_range_end.begin(),
     d_ignore_vals   = d_ignore_vals.begin(),
     d_columns_data  = d_columns_data.begin()] __device__(size_type i) {
      switch (node_categories[i]) {
        case NC_STRUCT: set_bit(d_columns_data[col_ids[i]].validity, row_offsets[i]); break;
        case NC_LIST: set_bit(d_columns_data[col_ids[i]].validity, row_offsets[i]); break;
        case NC_VAL:
        case NC_STR:
          if (d_ignore_vals[col_ids[i]]) break;
          set_bit(d_columns_data[col_ids[i]].validity, row_offsets[i]);
          d_columns_data[col_ids[i]].string_offsets[row_offsets[i]] = range_begin[i];
          d_columns_data[col_ids[i]].string_lengths[row_offsets[i]] = range_end[i] - range_begin[i];
          break;
        default: break;
      }
    });

  // 4. scatter List offset
  //   sort_by_key {col_id}, {node_id}
  //   unique_copy_by_key {parent_node_id} {row_offset} to
  //   col[parent_col_id].child_offsets[row_offset[parent_node_id]]

  rmm::device_uvector<NodeIndexT> original_col_ids(col_ids.size(), stream);  // make a copy
  thrust::copy(rmm::exec_policy(stream), col_ids.begin(), col_ids.end(), original_col_ids.begin());
  rmm::device_uvector<size_type> node_ids(row_offsets.size(), stream);
  thrust::sequence(rmm::exec_policy(stream), node_ids.begin(), node_ids.end());
  thrust::stable_sort_by_key(
    rmm::exec_policy(stream), col_ids.begin(), col_ids.end(), node_ids.begin());

  auto ordered_parent_node_ids =
    thrust::make_permutation_iterator(tree.parent_node_ids.begin(), node_ids.begin());
  auto ordered_row_offsets =
    thrust::make_permutation_iterator(row_offsets.begin(), node_ids.begin());
  thrust::for_each_n(
    rmm::exec_policy(stream),
    thrust::counting_iterator<size_type>(0),
    col_ids.size(),
    [num_nodes = col_ids.size(),
     ordered_parent_node_ids,
     ordered_row_offsets,
     original_col_ids = original_col_ids.begin(),
     col_ids          = col_ids.begin(),
     row_offsets      = row_offsets.begin(),
     node_categories  = tree.node_categories.begin(),
     d_columns_data   = d_columns_data.begin()] __device__(size_type i) {
      auto parent_node_id = ordered_parent_node_ids[i];
      if (parent_node_id != parent_node_sentinel and node_categories[parent_node_id] == NC_LIST) {
        // unique item
        if (i == 0 or
            (col_ids[i - 1] != col_ids[i] or ordered_parent_node_ids[i - 1] != parent_node_id)) {
          // scatter to list_offset
          d_columns_data[original_col_ids[parent_node_id]]
            .child_offsets[row_offsets[parent_node_id]] = ordered_row_offsets[i];
        }
        // TODO: verify if this code is right. check with more test cases.
        if (i == num_nodes - 1 or
            (col_ids[i] != col_ids[i + 1] or ordered_parent_node_ids[i + 1] != parent_node_id)) {
          // last value of list child_offset is its size.
          d_columns_data[original_col_ids[parent_node_id]]
            .child_offsets[row_offsets[parent_node_id] + 1] = ordered_row_offsets[i] + 1;
        }
      }
    });

  // restore col_ids, TODO is this required?
  // thrust::copy(
  //   rmm::exec_policy(stream), original_col_ids.begin(), original_col_ids.end(), col_ids.begin());

  // 5. scan on offsets.
  for (auto& [id, col_ref] : columns) {
    auto& col = col_ref.get();
    if (col.type == json_col_t::StringColumn) {
      thrust::inclusive_scan(rmm::exec_policy(stream),
                             col.string_offsets.begin(),
                             col.string_offsets.end(),
                             col.string_offsets.begin(),
                             thrust::maximum<json_column::row_offset_t>{});
    } else if (col.type == json_col_t::ListColumn) {
      thrust::inclusive_scan(rmm::exec_policy(stream),
                             col.child_offsets.begin(),
                             col.child_offsets.end(),
                             col.child_offsets.begin(),
                             thrust::maximum<json_column::row_offset_t>{});
    }
  }
}

/**
 * @brief Retrieves the parse_options to be used for type inference and type casting
 *
 * @param options The reader options to influence the relevant type inference and type casting
 * options
 */
cudf::io::parse_options parsing_options(cudf::io::json_reader_options const& options);

std::pair<std::unique_ptr<column>, std::vector<column_name_info>> device_json_column_to_cudf_column(
  device_json_column& json_col,
  device_span<SymbolT const> d_input,
  cudf::io::json_reader_options const& options,
  std::optional<schema_element> schema,
  rmm::cuda_stream_view stream,
  rmm::mr::device_memory_resource* mr)
{
  CUDF_FUNC_RANGE();
  auto make_validity =
    [stream](device_json_column& json_col) -> std::pair<rmm::device_buffer, size_type> {
    CUDF_EXPECTS(json_col.validity.size() >= bitmask_allocation_size_bytes(json_col.num_rows),
                 "valid_count is too small");
    auto null_count =
      cudf::detail::null_count(json_col.validity.data(), 0, json_col.num_rows, stream);
    // full null_mask is always required for parse_data
    return {json_col.validity.release(), null_count};
    // Note: json_col modified here, moves this memory
  };

  auto get_child_schema = [schema](auto child_name) -> std::optional<schema_element> {
    if (schema.has_value()) {
      auto const result = schema.value().child_types.find(child_name);
      if (result != std::end(schema.value().child_types)) { return result->second; }
    }
    return {};
  };

  switch (json_col.type) {
    case json_col_t::StringColumn: {
      // move string_offsets to GPU and transform to string column
      auto const col_size      = json_col.string_offsets.size();
      using char_length_pair_t = thrust::pair<const char*, size_type>;
      CUDF_EXPECTS(json_col.string_offsets.size() == json_col.string_lengths.size(),
                   "string offset, string length mismatch");
      rmm::device_uvector<char_length_pair_t> d_string_data(col_size, stream);
      // TODO how about directly storing pair<char*, size_t> in json_column?
      auto offset_length_it =
        thrust::make_zip_iterator(json_col.string_offsets.begin(), json_col.string_lengths.begin());
      // Prepare iterator that returns (string_offset, string_length)-pairs needed by inference
      auto string_ranges_it =
        thrust::make_transform_iterator(offset_length_it, [] __device__(auto ip) {
          return thrust::pair<json_column::row_offset_t, std::size_t>{
            thrust::get<0>(ip), static_cast<std::size_t>(thrust::get<1>(ip))};
        });

      // Prepare iterator that returns (string_ptr, string_length)-pairs needed by type conversion
      auto string_spans_it = thrust::make_transform_iterator(
        offset_length_it, [data = d_input.data()] __device__(auto ip) {
          return thrust::pair<const char*, std::size_t>{
            data + thrust::get<0>(ip), static_cast<std::size_t>(thrust::get<1>(ip))};
        });

      data_type target_type{};

      if (schema.has_value()) {
#ifdef NJP_DEBUG_PRINT
        std::cout << "-> explicit type: "
                  << (schema.has_value() ? std::to_string(static_cast<int>(schema->type.id()))
                                         : "n/a");
#endif
        target_type = schema.value().type;
      }
      // Infer column type, if we don't have an explicit type for it
      else {
        target_type = cudf::io::detail::infer_data_type(
          parsing_options(options).json_view(), d_input, string_ranges_it, col_size, stream);
      }
      // Convert strings to the inferred data type
      auto col = experimental::detail::parse_data(string_spans_it,
                                                  col_size,
                                                  target_type,
                                                  make_validity(json_col).first,
                                                  parsing_options(options).view(),
                                                  stream,
                                                  mr);

      // Reset nullable if we do not have nulls
      // This is to match the existing JSON reader's behaviour:
      // - Non-string columns will always be returned as nullable
      // - String columns will be returned as nullable, iff there's at least one null entry
      if (target_type.id() == type_id::STRING and col->null_count() == 0) {
        col->set_null_mask(rmm::device_buffer{0, stream, mr}, 0);
      }

      // For string columns return ["offsets", "char"] schema
      if (target_type.id() == type_id::STRING) {
        return {std::move(col), {{"offsets"}, {"chars"}}};
      }
      // Non-string leaf-columns (e.g., numeric) do not have child columns in the schema
      return {std::move(col), {}};
    }
    case json_col_t::StructColumn: {
      std::vector<std::unique_ptr<column>> child_columns;
      std::vector<column_name_info> column_names{};
      size_type num_rows{json_col.num_rows};
      // Create children columns
      for (auto const& col_name : json_col.column_order) {
        auto const& col = json_col.child_columns.find(col_name);
        column_names.emplace_back(col->first);
        auto& child_col            = col->second;
        auto [child_column, names] = device_json_column_to_cudf_column(
          child_col, d_input, options, get_child_schema(col_name), stream, mr);
        CUDF_EXPECTS(num_rows == child_column->size(),
                     "All children columns must have the same size");
        child_columns.push_back(std::move(child_column));
        column_names.back().children = names;
      }
      auto [result_bitmask, null_count] = make_validity(json_col);
      return {
        make_structs_column(
          num_rows, std::move(child_columns), null_count, std::move(result_bitmask), stream, mr),
        column_names};
    }
    case json_col_t::ListColumn: {
      size_type num_rows = json_col.child_offsets.size() - 1;
      std::vector<column_name_info> column_names{};
      column_names.emplace_back("offsets");
      column_names.emplace_back(
        json_col.child_columns.empty() ? list_child_name : json_col.child_columns.begin()->first);

      // Note: json_col modified here, reuse the memory
      auto offsets_column = std::make_unique<column>(
        data_type{type_id::INT32}, num_rows + 1, json_col.child_offsets.release());
      // Create children column
      auto [child_column, names] =
        json_col.child_columns.empty()
          ? std::pair<std::unique_ptr<column>,
                      std::vector<column_name_info>>{std::make_unique<column>(), {}}
          : device_json_column_to_cudf_column(
              json_col.child_columns.begin()->second,
              d_input,
              options,
              get_child_schema(json_col.child_columns.begin()->first),
              stream,
              mr);
      column_names.back().children      = names;
      auto [result_bitmask, null_count] = make_validity(json_col);
      return {make_lists_column(num_rows,
                                std::move(offsets_column),
                                std::move(child_column),
                                null_count,
                                std::move(result_bitmask),
                                stream,
                                mr),
              std::move(column_names)};
    }
    default: CUDF_FAIL("Unsupported column type"); break;
  }
}

table_with_metadata device_parse_nested_json(device_span<SymbolT const> d_input,
                                             cudf::io::json_reader_options const& options,
                                             rmm::cuda_stream_view stream,
                                             rmm::mr::device_memory_resource* mr)
{
  CUDF_FUNC_RANGE();

  auto gpu_tree = [&]() {
    // Parse the JSON and get the token stream
    const auto [tokens_gpu, token_indices_gpu] = get_token_stream(d_input, options, stream);
    // gpu tree generation
    return get_tree_representation(tokens_gpu, token_indices_gpu, stream);
  }();  // IILE used to free memory of token data.
#ifdef NJP_DEBUG_PRINT
  auto h_input = cudf::detail::make_host_vector_async(d_input, stream);
  print_tree(h_input, gpu_tree, stream);
#endif

  auto [gpu_col_id, gpu_row_offsets] = records_orient_tree_traversal(d_input, gpu_tree, stream);

  device_json_column root_column(stream, mr);
  root_column.type = json_col_t::ListColumn;
  root_column.child_offsets.resize(2, stream);
  thrust::fill(rmm::exec_policy(stream),
               root_column.child_offsets.begin(),
               root_column.child_offsets.end(),
               0);

  // Get internal JSON column
  make_device_json_column(d_input, gpu_tree, gpu_col_id, gpu_row_offsets, root_column, stream, mr);

  // data_root refers to the root column of the data represented by the given JSON string
  auto& data_root =
    options.is_enabled_lines() ? root_column : root_column.child_columns.begin()->second;

  // Zero row entries
  if (data_root.type == json_col_t::ListColumn && data_root.child_columns.size() == 0) {
    return table_with_metadata{std::make_unique<table>(std::vector<std::unique_ptr<column>>{}),
                               {{}, std::vector<column_name_info>{}}};
  }

  // Verify that we were in fact given a list of structs (or in JSON speech: an array of objects)
  auto constexpr single_child_col_count = 1;
  CUDF_EXPECTS(data_root.type == json_col_t::ListColumn and
                 data_root.child_columns.size() == single_child_col_count and
                 data_root.child_columns.begin()->second.type == json_col_t::StructColumn,
               "Currently the nested JSON parser only supports an array of (nested) objects");

  // Slice off the root list column, which has only a single row that contains all the structs
  auto& root_struct_col = data_root.child_columns.begin()->second;

  // Initialize meta data to be populated while recursing through the tree of columns
  std::vector<std::unique_ptr<column>> out_columns;
  std::vector<column_name_info> out_column_names;

  // Iterate over the struct's child columns and convert to cudf column
  size_type column_index = 0;
  for (auto const& col_name : root_struct_col.column_order) {
    auto& json_col = root_struct_col.child_columns.find(col_name)->second;
    // Insert this columns name into the schema
    out_column_names.emplace_back(col_name);

    std::optional<schema_element> child_schema_element = std::visit(
      cudf::detail::visitor_overload{
        [column_index](const std::vector<data_type>& user_dtypes) -> std::optional<schema_element> {
          return (static_cast<std::size_t>(column_index) < user_dtypes.size())
                   ? std::optional<schema_element>{{user_dtypes[column_index]}}
                   : std::optional<schema_element>{};
        },
        [col_name](
          std::map<std::string, data_type> const& user_dtypes) -> std::optional<schema_element> {
          return (user_dtypes.find(col_name) != std::end(user_dtypes))
                   ? std::optional<schema_element>{{user_dtypes.find(col_name)->second}}
                   : std::optional<schema_element>{};
        },
        [col_name](std::map<std::string, schema_element> const& user_dtypes)
          -> std::optional<schema_element> {
          return (user_dtypes.find(col_name) != std::end(user_dtypes))
                   ? user_dtypes.find(col_name)->second
                   : std::optional<schema_element>{};
        }},
      options.get_dtypes());
#ifdef NJP_DEBUG_PRINT
    auto debug_schema_print = [](auto ret) {
      std::cout << ", type id: "
                << (ret.has_value() ? std::to_string(static_cast<int>(ret->type.id())) : "n/a")
                << ", with " << (ret.has_value() ? ret->child_types.size() : 0) << " children"
                << "\n";
    };
    std::visit(
      cudf::detail::visitor_overload{[column_index](const std::vector<data_type>&) {
                                       std::cout << "Column by index: #" << column_index;
                                     },
                                     [col_name](std::map<std::string, data_type> const&) {
                                       std::cout << "Column by flat name: '" << col_name;
                                     },
                                     [col_name](std::map<std::string, schema_element> const&) {
                                       std::cout << "Column by nested name: #" << col_name;
                                     }},
      options.get_dtypes());
    debug_schema_print(child_schema_element);
#endif

    // Get this JSON column's cudf column and schema info, (modifies json_col)
    auto [cudf_col, col_name_info] = device_json_column_to_cudf_column(
      json_col, d_input, options, child_schema_element, stream, mr);

    out_column_names.back().children = std::move(col_name_info);
    out_columns.emplace_back(std::move(cudf_col));

    column_index++;
  }

  return table_with_metadata{std::make_unique<table>(std::move(out_columns)),
                             {{}, out_column_names}};
}

table_with_metadata device_parse_nested_json(host_span<SymbolT const> input,
                                             cudf::io::json_reader_options const& options,
                                             rmm::cuda_stream_view stream,
                                             rmm::mr::device_memory_resource* mr)
{
  CUDF_FUNC_RANGE();

  // Allocate device memory for the JSON input & copy over to device
  rmm::device_uvector<SymbolT> d_input = cudf::detail::make_device_uvector_async(input, stream);

  return device_parse_nested_json(device_span<SymbolT const>{d_input}, options, stream, mr);
}
}  // namespace detail
}  // namespace cudf::io::json
