#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2022-2024, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "io/utilities/parsing_utils.cuh"
#include "io/utilities/string_parsing.hpp"
#include "nested_json.hpp"

#include <cudf/column/column_factories.hpp>
#include <cudf/detail/null_mask.hpp>
#include <cudf/detail/nvtx/ranges.hpp>
#include <cudf/detail/utilities/vector_factories.hpp>
#include <cudf/detail/utilities/visitor_overload.hpp>
#include <cudf/strings/strings_column_view.hpp>
#include <cudf/types.hpp>
#include <cudf/utilities/error.hpp>
#include <cudf/utilities/span.hpp>

#include <rmm/cuda_stream_view.hpp>
#include <rmm/device_uvector.hpp>
#include <rmm/exec_policy.hpp>

#include <cuda/atomic>
#include <cuda/functional>
#include <thrust/count.h>
#include <thrust/for_each.h>
#include <thrust/functional.h>
#include <thrust/gather.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/iterator/discard_iterator.h>
#include <thrust/iterator/permutation_iterator.h>
#include <thrust/iterator/zip_iterator.h>
#include <thrust/reduce.h>
#include <thrust/scan.h>
#include <thrust/sort.h>
#include <thrust/transform.h>
#include <thrust/unique.h>

#include <algorithm>
#include <cstdint>

namespace cudf::io::json::detail {

// DEBUG prints
auto to_cat = [](auto v) -> std::string {
  switch (v) {
    case NC_STRUCT: return " S";
    case NC_LIST: return " L";
    case NC_STR: return " \"";
    case NC_VAL: return " V";
    case NC_FN: return " F";
    case NC_ERR: return "ER";
    default: return "UN";
  };
};
auto to_int    = [](auto v) { return std::to_string(static_cast<int>(v)); };
auto print_vec = [](auto const& cpu, auto const name, auto converter) {
  for (auto const& v : cpu)
    printf("%3s,", converter(v).c_str());
  std::cout << name << std::endl;
};

void print_tree(host_span<SymbolT const> input,
                tree_meta_t const& d_gpu_tree,
                rmm::cuda_stream_view stream)
{
  print_vec(cudf::detail::make_std_vector_async(d_gpu_tree.node_categories, stream),
            "node_categories",
            to_cat);
  print_vec(cudf::detail::make_std_vector_async(d_gpu_tree.parent_node_ids, stream),
            "parent_node_ids",
            to_int);
  print_vec(
    cudf::detail::make_std_vector_async(d_gpu_tree.node_levels, stream), "node_levels", to_int);
  auto node_range_begin = cudf::detail::make_std_vector_async(d_gpu_tree.node_range_begin, stream);
  auto node_range_end   = cudf::detail::make_std_vector_async(d_gpu_tree.node_range_end, stream);
  print_vec(node_range_begin, "node_range_begin", to_int);
  print_vec(node_range_end, "node_range_end", to_int);
  for (int i = 0; i < int(node_range_begin.size()); i++) {
    printf("%3s ",
           std::string(input.data() + node_range_begin[i], node_range_end[i] - node_range_begin[i])
             .c_str());
  }
  printf(" (JSON)\n");
}

/**
 * @brief Reduces node tree representation to column tree representation.
 *
 * @param tree Node tree representation of JSON string
 * @param original_col_ids Column ids of nodes
 * @param sorted_col_ids Sorted column ids of nodes
 * @param ordered_node_ids Node ids of nodes sorted by column ids
 * @param row_offsets Row offsets of nodes
 * @param is_array_of_arrays Whether the tree is an array of arrays
 * @param row_array_parent_col_id Column id of row array, if is_array_of_arrays is true
 * @param stream CUDA stream used for device memory operations and kernel launches
 * @return A tuple of column tree representation of JSON string, column ids of columns, and
 * max row offsets of columns
 */
std::tuple<tree_meta_t, rmm::device_uvector<NodeIndexT>, rmm::device_uvector<size_type>>
reduce_to_column_tree(tree_meta_t& tree,
                      device_span<NodeIndexT> original_col_ids,
                      device_span<NodeIndexT> sorted_col_ids,
                      device_span<NodeIndexT> ordered_node_ids,
                      device_span<size_type> row_offsets,
                      bool is_array_of_arrays,
                      NodeIndexT const row_array_parent_col_id,
                      rmm::cuda_stream_view stream)
{
  CUDF_FUNC_RANGE();
  // 1. column count for allocation
  auto const num_columns =
    thrust::unique_count(rmm::exec_policy(stream), sorted_col_ids.begin(), sorted_col_ids.end());

  // 2. reduce_by_key {col_id}, {row_offset}, max.
  rmm::device_uvector<NodeIndexT> unique_col_ids(num_columns, stream);
  rmm::device_uvector<size_type> max_row_offsets(num_columns, stream);
  auto ordered_row_offsets =
    thrust::make_permutation_iterator(row_offsets.begin(), ordered_node_ids.begin());
  thrust::reduce_by_key(rmm::exec_policy(stream),
                        sorted_col_ids.begin(),
                        sorted_col_ids.end(),
                        ordered_row_offsets,
                        unique_col_ids.begin(),
                        max_row_offsets.begin(),
                        thrust::equal_to<size_type>(),
                        thrust::maximum<size_type>());

  // 3. reduce_by_key {col_id}, {node_categories} - custom opp (*+v=*, v+v=v, *+#=E)
  rmm::device_uvector<NodeT> column_categories(num_columns, stream);
  thrust::reduce_by_key(
    rmm::exec_policy(stream),
    sorted_col_ids.begin(),
    sorted_col_ids.end(),
    thrust::make_permutation_iterator(tree.node_categories.begin(), ordered_node_ids.begin()),
    unique_col_ids.begin(),
    column_categories.begin(),
    thrust::equal_to<size_type>(),
    [] __device__(NodeT type_a, NodeT type_b) -> NodeT {
      auto is_a_leaf = (type_a == NC_VAL || type_a == NC_STR);
      auto is_b_leaf = (type_b == NC_VAL || type_b == NC_STR);
      // (v+v=v, *+*=*,  *+v=*, *+#=E, NESTED+VAL=NESTED)
      // *+*=*, v+v=v
      if (type_a == type_b) {
        return type_a;
      } else if (is_a_leaf) {
        // *+v=*, N+V=N
        // STRUCT/LIST + STR/VAL = STRUCT/LIST, STR/VAL + FN = ERR, STR/VAL + STR = STR
        return type_b == NC_FN ? NC_ERR : (is_b_leaf ? NC_STR : type_b);
      } else if (is_b_leaf) {
        return type_a == NC_FN ? NC_ERR : (is_a_leaf ? NC_STR : type_a);
      }
      // *+#=E
      return NC_ERR;
    });

  // 4. unique_copy parent_node_ids, ranges
  rmm::device_uvector<TreeDepthT> column_levels(0, stream);  // not required
  rmm::device_uvector<NodeIndexT> parent_col_ids(num_columns, stream);
  rmm::device_uvector<SymbolOffsetT> col_range_begin(num_columns, stream);  // Field names
  rmm::device_uvector<SymbolOffsetT> col_range_end(num_columns, stream);
  rmm::device_uvector<size_type> unique_node_ids(num_columns, stream);
  thrust::unique_by_key_copy(rmm::exec_policy(stream),
                             sorted_col_ids.begin(),
                             sorted_col_ids.end(),
                             ordered_node_ids.begin(),
                             thrust::make_discard_iterator(),
                             unique_node_ids.begin());
  thrust::copy_n(
    rmm::exec_policy(stream),
    thrust::make_zip_iterator(
      thrust::make_permutation_iterator(tree.parent_node_ids.begin(), unique_node_ids.begin()),
      thrust::make_permutation_iterator(tree.node_range_begin.begin(), unique_node_ids.begin()),
      thrust::make_permutation_iterator(tree.node_range_end.begin(), unique_node_ids.begin())),
    unique_node_ids.size(),
    thrust::make_zip_iterator(
      parent_col_ids.begin(), col_range_begin.begin(), col_range_end.begin()));

  // convert parent_node_ids to parent_col_ids
  thrust::transform(
    rmm::exec_policy(stream),
    parent_col_ids.begin(),
    parent_col_ids.end(),
    parent_col_ids.begin(),
    [col_ids = original_col_ids.begin()] __device__(auto parent_node_id) -> size_type {
      return parent_node_id == parent_node_sentinel ? parent_node_sentinel
                                                    : col_ids[parent_node_id];
    });

  // condition is true if parent is not a list, or sentinel/root
  // Special case to return true if parent is a list and is_array_of_arrays is true
  auto is_non_list_parent = [column_categories = column_categories.begin(),
                             is_array_of_arrays,
                             row_array_parent_col_id] __device__(auto parent_col_id) -> bool {
    return !(parent_col_id == parent_node_sentinel ||
             column_categories[parent_col_id] == NC_LIST &&
               (!is_array_of_arrays || parent_col_id != row_array_parent_col_id));
  };
  // Mixed types in List children go to different columns,
  // so all immediate children of list column should have same max_row_offsets.
  //   create list's children max_row_offsets array. (initialize to zero)
  //   atomicMax on  children max_row_offsets array.
  //   gather the max_row_offsets from children row offset array.
  {
    rmm::device_uvector<NodeIndexT> list_parents_children_max_row_offsets(num_columns, stream);
    thrust::fill(rmm::exec_policy(stream),
                 list_parents_children_max_row_offsets.begin(),
                 list_parents_children_max_row_offsets.end(),
                 0);
    thrust::for_each(rmm::exec_policy(stream),
                     unique_col_ids.begin(),
                     unique_col_ids.end(),
                     [column_categories = column_categories.begin(),
                      parent_col_ids    = parent_col_ids.begin(),
                      max_row_offsets   = max_row_offsets.begin(),
                      list_parents_children_max_row_offsets =
                        list_parents_children_max_row_offsets.begin()] __device__(auto col_id) {
                       auto parent_col_id = parent_col_ids[col_id];
                       if (parent_col_id != parent_node_sentinel and
                           column_categories[parent_col_id] == node_t::NC_LIST) {
                         cuda::atomic_ref<NodeIndexT, cuda::thread_scope_device> ref{
                           *(list_parents_children_max_row_offsets + parent_col_id)};
                         ref.fetch_max(max_row_offsets[col_id], cuda::std::memory_order_relaxed);
                       }
                     });
    thrust::gather_if(
      rmm::exec_policy(stream),
      parent_col_ids.begin(),
      parent_col_ids.end(),
      parent_col_ids.begin(),
      list_parents_children_max_row_offsets.begin(),
      max_row_offsets.begin(),
      [column_categories = column_categories.begin()] __device__(size_type parent_col_id) {
        return parent_col_id != parent_node_sentinel and
               column_categories[parent_col_id] == node_t::NC_LIST;
      });
  }

  // copy lists' max_row_offsets to children.
  // all structs should have same size.
  thrust::transform_if(
    rmm::exec_policy(stream),
    unique_col_ids.begin(),
    unique_col_ids.end(),
    max_row_offsets.begin(),
    [column_categories = column_categories.begin(),
     is_non_list_parent,
     parent_col_ids  = parent_col_ids.begin(),
     max_row_offsets = max_row_offsets.begin()] __device__(size_type col_id) {
      auto parent_col_id = parent_col_ids[col_id];
      // condition is true if parent is not a list, or sentinel/root
      while (is_non_list_parent(parent_col_id)) {
        col_id        = parent_col_id;
        parent_col_id = parent_col_ids[parent_col_id];
      }
      return max_row_offsets[col_id];
    },
    [column_categories = column_categories.begin(),
     is_non_list_parent,
     parent_col_ids = parent_col_ids.begin()] __device__(size_type col_id) {
      auto parent_col_id = parent_col_ids[col_id];
      // condition is true if parent is not a list, or sentinel/root
      return is_non_list_parent(parent_col_id);
    });

  // For Struct and List (to avoid copying entire strings when mixed type as string is enabled)
  thrust::transform_if(
    rmm::exec_policy(stream),
    col_range_begin.begin(),
    col_range_begin.end(),
    column_categories.begin(),
    col_range_end.begin(),
    [] __device__(auto i) { return i + 1; },
    [] __device__(NodeT type) { return type == NC_STRUCT || type == NC_LIST; });

  return std::tuple{tree_meta_t{std::move(column_categories),
                                std::move(parent_col_ids),
                                std::move(column_levels),
                                std::move(col_range_begin),
                                std::move(col_range_end)},
                    std::move(unique_col_ids),
                    std::move(max_row_offsets)};
}

/**
 * @brief Get the column indices for the values column for array of arrays rows
 *
 * @param row_array_children_level The level of the row array's children
 * @param d_tree The tree metadata
 * @param col_ids The column ids
 * @param num_columns The number of columns
 * @param stream The stream to use
 * @return The value columns' indices
 */
rmm::device_uvector<NodeIndexT> get_values_column_indices(TreeDepthT const row_array_children_level,
                                                          tree_meta_t const& d_tree,
                                                          device_span<NodeIndexT> col_ids,
                                                          size_type const num_columns,
                                                          rmm::cuda_stream_view stream)
{
  CUDF_FUNC_RANGE();
  auto [level2_nodes, level2_indices] = get_array_children_indices(
    row_array_children_level, d_tree.node_levels, d_tree.parent_node_ids, stream);
  auto col_id_location = thrust::make_permutation_iterator(col_ids.begin(), level2_nodes.begin());
  rmm::device_uvector<NodeIndexT> values_column_indices(num_columns, stream);
  thrust::scatter(rmm::exec_policy(stream),
                  level2_indices.begin(),
                  level2_indices.end(),
                  col_id_location,
                  values_column_indices.begin());
  return values_column_indices;
}

/**
 * @brief Copies strings specified by pair of begin, end offsets to host vector of strings.
 *
 * @param input String device buffer
 * @param node_range_begin Begin offset of the strings
 * @param node_range_end End offset of the strings
 * @param stream CUDA stream
 * @return Vector of strings
 */
std::vector<std::string> copy_strings_to_host(device_span<SymbolT const> input,
                                              device_span<SymbolOffsetT const> node_range_begin,
                                              device_span<SymbolOffsetT const> node_range_end,
                                              rmm::cuda_stream_view stream)
{
  CUDF_FUNC_RANGE();
  auto const num_strings = node_range_begin.size();
  rmm::device_uvector<size_type> string_offsets(num_strings, stream);
  rmm::device_uvector<size_type> string_lengths(num_strings, stream);
  auto d_offset_pairs = thrust::make_zip_iterator(node_range_begin.begin(), node_range_end.begin());
  thrust::transform(rmm::exec_policy(stream),
                    d_offset_pairs,
                    d_offset_pairs + num_strings,
                    thrust::make_zip_iterator(string_offsets.begin(), string_lengths.begin()),
                    [] __device__(auto const& offsets) {
                      // Note: first character for non-field columns
                      return thrust::make_tuple(
                        static_cast<size_type>(thrust::get<0>(offsets)),
                        static_cast<size_type>(thrust::get<1>(offsets) - thrust::get<0>(offsets)));
                    });

  cudf::io::parse_options_view options_view{};
  options_view.quotechar  = '\0';  // no quotes
  options_view.keepquotes = true;
  auto d_offset_length_it =
    thrust::make_zip_iterator(string_offsets.begin(), string_lengths.begin());
  auto d_column_names = parse_data(input.data(),
                                   d_offset_length_it,
                                   num_strings,
                                   data_type{type_id::STRING},
                                   rmm::device_buffer{},
                                   0,
                                   options_view,
                                   stream,
                                   rmm::mr::get_current_device_resource());
  auto to_host        = [stream](auto const& col) {
    if (col.is_empty()) return std::vector<std::string>{};
    auto const scv     = cudf::strings_column_view(col);
    auto const h_chars = cudf::detail::make_std_vector_sync<char>(
      cudf::device_span<char const>(scv.chars_begin(stream), scv.chars_size(stream)), stream);
    auto const h_offsets = cudf::detail::make_std_vector_sync(
      cudf::device_span<cudf::size_type const>(scv.offsets().data<cudf::size_type>() + scv.offset(),
                                               scv.size() + 1),
      stream);

    // build std::string vector from chars and offsets
    std::vector<std::string> host_data;
    host_data.reserve(col.size());
    std::transform(
      std::begin(h_offsets),
      std::end(h_offsets) - 1,
      std::begin(h_offsets) + 1,
      std::back_inserter(host_data),
      [&](auto start, auto end) { return std::string(h_chars.data() + start, end - start); });
    return host_data;
  };
  return to_host(d_column_names->view());
}

/**
 * @brief Checks if all strings in each string column in the tree are nulls.
 * For non-string columns, it's set as true. If any of rows in a string column is false, it's set as
 * false.
 *
 * @param input Input JSON string device data
 * @param d_column_tree column tree representation of JSON string
 * @param tree Node tree representation of the JSON string
 * @param col_ids Column ids of the nodes in the tree
 * @param options Parsing options specifying the parsing behaviour
 * @param stream CUDA stream used for device memory operations and kernel launches
 * @return Array of bytes where each byte indicate if it is all nulls string column.
 */
rmm::device_uvector<uint8_t> is_all_nulls_each_column(device_span<SymbolT const> input,
                                                      tree_meta_t const& d_column_tree,
                                                      tree_meta_t const& tree,
                                                      device_span<NodeIndexT> col_ids,
                                                      cudf::io::json_reader_options const& options,
                                                      rmm::cuda_stream_view stream)
{
  auto const num_nodes = col_ids.size();
  auto const num_cols  = d_column_tree.node_categories.size();
  rmm::device_uvector<uint8_t> is_all_nulls(num_cols, stream);
  thrust::fill(rmm::exec_policy(stream), is_all_nulls.begin(), is_all_nulls.end(), true);

  auto parse_opt = parsing_options(options, stream);
  thrust::for_each_n(
    rmm::exec_policy(stream),
    thrust::counting_iterator<size_type>(0),
    num_nodes,
    [options           = parse_opt.view(),
     data              = input.data(),
     column_categories = d_column_tree.node_categories.begin(),
     col_ids           = col_ids.begin(),
     range_begin       = tree.node_range_begin.begin(),
     range_end         = tree.node_range_end.begin(),
     is_all_nulls      = is_all_nulls.begin()] __device__(size_type i) {
      auto const node_category = column_categories[col_ids[i]];
      if (node_category == NC_STR or node_category == NC_VAL) {
        auto const is_null_literal = serialized_trie_contains(
          options.trie_na,
          {data + range_begin[i], static_cast<size_t>(range_end[i] - range_begin[i])});
        if (!is_null_literal) is_all_nulls[col_ids[i]] = false;
      }
    });
  return is_all_nulls;
}

/**
 * @brief Holds member data pointers of `d_json_column`
 *
 */
struct json_column_data {
  using row_offset_t = json_column::row_offset_t;
  row_offset_t* string_offsets;
  row_offset_t* string_lengths;
  row_offset_t* child_offsets;
  bitmask_type* validity;
};

/**
 * @brief Constructs `d_json_column` from node tree representation
 * Newly constructed columns are insert into `root`'s children.
 * `root` must be a list type.
 *
 * @param input Input JSON string device data
 * @param tree Node tree representation of the JSON string
 * @param col_ids Column ids of the nodes in the tree
 * @param row_offsets Row offsets of the nodes in the tree
 * @param root Root node of the `d_json_column` tree
 * @param is_array_of_arrays Whether the tree is an array of arrays
 * @param options Parsing options specifying the parsing behaviour
 * options affecting behaviour are
 *   is_enabled_lines: Whether the input is a line-delimited JSON
 *   is_enabled_mixed_types_as_string: Whether to enable reading mixed types as string
 * @param stream CUDA stream used for device memory operations and kernel launches
 * @param mr Device memory resource used to allocate the device memory
 * of child_offets and validity members of `d_json_column`
 */
void make_device_json_column(device_span<SymbolT const> input,
                             tree_meta_t& tree,
                             device_span<NodeIndexT> col_ids,
                             device_span<size_type> row_offsets,
                             device_json_column& root,
                             bool is_array_of_arrays,
                             cudf::io::json_reader_options const& options,
                             rmm::cuda_stream_view stream,
                             rmm::mr::device_memory_resource* mr)
{
  CUDF_FUNC_RANGE();

  bool const is_enabled_lines                 = options.is_enabled_lines();
  bool const is_enabled_mixed_types_as_string = options.is_enabled_mixed_types_as_string();
  auto const num_nodes                        = col_ids.size();
  rmm::device_uvector<NodeIndexT> sorted_col_ids(col_ids.size(), stream);  // make a copy
  thrust::copy(rmm::exec_policy(stream), col_ids.begin(), col_ids.end(), sorted_col_ids.begin());

  // sort by {col_id} on {node_ids} stable
  rmm::device_uvector<NodeIndexT> node_ids(col_ids.size(), stream);
  thrust::sequence(rmm::exec_policy(stream), node_ids.begin(), node_ids.end());
  thrust::stable_sort_by_key(
    rmm::exec_policy(stream), sorted_col_ids.begin(), sorted_col_ids.end(), node_ids.begin());

  NodeIndexT const row_array_parent_col_id = [&]() {
    if (!is_array_of_arrays) return parent_node_sentinel;
    auto const list_node_index = is_enabled_lines ? 0 : 1;
    NodeIndexT value;
    CUDF_CUDA_TRY(hipMemcpyAsync(&value,
                                  col_ids.data() + list_node_index,
                                  sizeof(NodeIndexT),
                                  hipMemcpyDefault,
                                  stream.value()));
    stream.synchronize();
    return value;
  }();

  // 1. gather column information.
  auto [d_column_tree, d_unique_col_ids, d_max_row_offsets] =
    reduce_to_column_tree(tree,
                          col_ids,
                          sorted_col_ids,
                          node_ids,
                          row_offsets,
                          is_array_of_arrays,
                          row_array_parent_col_id,
                          stream);
  auto num_columns    = d_unique_col_ids.size();
  auto unique_col_ids = cudf::detail::make_std_vector_async(d_unique_col_ids, stream);
  auto column_categories =
    cudf::detail::make_std_vector_async(d_column_tree.node_categories, stream);
  auto column_parent_ids =
    cudf::detail::make_std_vector_async(d_column_tree.parent_node_ids, stream);
  auto column_range_beg =
    cudf::detail::make_std_vector_async(d_column_tree.node_range_begin, stream);
  auto max_row_offsets = cudf::detail::make_std_vector_async(d_max_row_offsets, stream);
  std::vector<std::string> column_names = copy_strings_to_host(
    input, d_column_tree.node_range_begin, d_column_tree.node_range_end, stream);
  // array of arrays column names
  if (is_array_of_arrays) {
    TreeDepthT const row_array_children_level = is_enabled_lines ? 1 : 2;
    auto values_column_indices =
      get_values_column_indices(row_array_children_level, tree, col_ids, num_columns, stream);
    auto h_values_column_indices =
      cudf::detail::make_std_vector_async(values_column_indices, stream);
    std::transform(unique_col_ids.begin(),
                   unique_col_ids.end(),
                   column_names.begin(),
                   column_names.begin(),
                   [&h_values_column_indices, &column_parent_ids, row_array_parent_col_id](
                     auto col_id, auto name) mutable {
                     return column_parent_ids[col_id] == row_array_parent_col_id
                              ? std::to_string(h_values_column_indices[col_id])
                              : name;
                   });
  }

  auto to_json_col_type = [](auto category) {
    switch (category) {
      case NC_STRUCT: return json_col_t::StructColumn;
      case NC_LIST: return json_col_t::ListColumn;
      case NC_STR: [[fallthrough]];
      case NC_VAL: return json_col_t::StringColumn;
      default: return json_col_t::Unknown;
    }
  };
  auto init_to_zero = [stream](auto& v) {
    thrust::uninitialized_fill(rmm::exec_policy(stream), v.begin(), v.end(), 0);
  };

  auto initialize_json_columns = [&](auto i, auto& col) {
    if (column_categories[i] == NC_ERR || column_categories[i] == NC_FN) {
      return;
    } else if (column_categories[i] == NC_VAL || column_categories[i] == NC_STR) {
      col.string_offsets.resize(max_row_offsets[i] + 1, stream);
      col.string_lengths.resize(max_row_offsets[i] + 1, stream);
      init_to_zero(col.string_offsets);
      init_to_zero(col.string_lengths);
    } else if (column_categories[i] == NC_LIST) {
      col.child_offsets.resize(max_row_offsets[i] + 2, stream);
      init_to_zero(col.child_offsets);
    }
    col.num_rows = max_row_offsets[i] + 1;
    col.validity =
      cudf::detail::create_null_mask(col.num_rows, cudf::mask_state::ALL_NULL, stream, mr);
    col.type = to_json_col_type(column_categories[i]);
  };

  auto reinitialize_as_string = [&](auto i, auto& col) {
    col.string_offsets.resize(max_row_offsets[i] + 1, stream);
    col.string_lengths.resize(max_row_offsets[i] + 1, stream);
    init_to_zero(col.string_offsets);
    init_to_zero(col.string_lengths);
    col.num_rows = max_row_offsets[i] + 1;
    col.validity =
      cudf::detail::create_null_mask(col.num_rows, cudf::mask_state::ALL_NULL, stream, mr);
    col.type = json_col_t::StringColumn;
    col.child_columns.clear();  // their references should be deleted too.
    col.column_order.clear();
  };

  // 2. generate nested columns tree and its device_memory
  // reorder unique_col_ids w.r.t. column_range_begin for order of column to be in field order.
  auto h_range_col_id_it =
    thrust::make_zip_iterator(column_range_beg.begin(), unique_col_ids.begin());
  std::sort(h_range_col_id_it, h_range_col_id_it + num_columns, [](auto const& a, auto const& b) {
    return thrust::get<0>(a) < thrust::get<0>(b);
  });

  std::vector<uint8_t> is_str_column_all_nulls{};
  if (is_enabled_mixed_types_as_string) {
    is_str_column_all_nulls = cudf::detail::make_std_vector_async(
      is_all_nulls_each_column(input, d_column_tree, tree, col_ids, options, stream), stream);
  }

  // use hash map because we may skip field name's col_ids
  std::unordered_map<NodeIndexT, std::reference_wrapper<device_json_column>> columns;
  // map{parent_col_id, child_col_name}> = child_col_id, used for null value column tracking
  std::map<std::pair<NodeIndexT, std::string>, NodeIndexT> mapped_columns;
  // find column_ids which are values, but should be ignored in validity
  std::vector<uint8_t> ignore_vals(num_columns, 0);
  std::vector<uint8_t> is_mixed_type_column(num_columns, 0);
  columns.try_emplace(parent_node_sentinel, std::ref(root));

  for (auto const this_col_id : unique_col_ids) {
    if (column_categories[this_col_id] == NC_ERR || column_categories[this_col_id] == NC_FN) {
      continue;
    }
    // Struct, List, String, Value
    std::string name   = "";
    auto parent_col_id = column_parent_ids[this_col_id];
    if (parent_col_id == parent_node_sentinel || column_categories[parent_col_id] == NC_LIST) {
      if (is_array_of_arrays && parent_col_id == row_array_parent_col_id) {
        name = column_names[this_col_id];
      } else {
        name = list_child_name;
      }
    } else if (column_categories[parent_col_id] == NC_FN) {
      auto field_name_col_id = parent_col_id;
      parent_col_id          = column_parent_ids[parent_col_id];
      name                   = column_names[field_name_col_id];
    } else {
      CUDF_FAIL("Unexpected parent column category");
    }

    if (parent_col_id != parent_node_sentinel && is_mixed_type_column[parent_col_id] == 1) {
      // if parent is mixed type column, ignore this column.
      is_mixed_type_column[this_col_id] = 1;
      ignore_vals[this_col_id]          = 1;
      continue;
    }
    // If the child is already found,
    // replace if this column is a nested column and the existing was a value column
    // ignore this column if this column is a value column and the existing was a nested column
    auto it = columns.find(parent_col_id);
    CUDF_EXPECTS(it != columns.end(), "Parent column not found");
    auto& parent_col = it->second.get();
    bool replaced    = false;
    if (mapped_columns.count({parent_col_id, name}) > 0) {
      auto const old_col_id = mapped_columns[{parent_col_id, name}];
      // If mixed type as string is enabled, make both of them strings and merge them.
      // All child columns will be ignored when parsing.
      if (is_enabled_mixed_types_as_string) {
        bool const is_mixed_type = [&]() {
          // If new or old is STR and they are all not null, make it mixed type, else ignore.
          if (column_categories[this_col_id] == NC_VAL ||
              column_categories[this_col_id] == NC_STR) {
            if (is_str_column_all_nulls[this_col_id]) return false;
          }
          if (column_categories[old_col_id] == NC_VAL || column_categories[old_col_id] == NC_STR) {
            if (is_str_column_all_nulls[old_col_id]) return false;
          }
          return true;
        }();
        if (is_mixed_type) {
          is_mixed_type_column[this_col_id] = 1;
          is_mixed_type_column[old_col_id]  = 1;
          // if old col type (not cat) is list or struct, replace with string.
          auto& col = columns.at(old_col_id).get();
          if (col.type == json_col_t::ListColumn or col.type == json_col_t::StructColumn) {
            reinitialize_as_string(old_col_id, col);
            // all its children (which are already inserted) are ignored later.
          }
          col.forced_as_string_column = true;
          columns.try_emplace(this_col_id, columns.at(old_col_id));
          continue;
        }
      }

      if (column_categories[this_col_id] == NC_VAL || column_categories[this_col_id] == NC_STR) {
        ignore_vals[this_col_id] = 1;
        continue;
      }
      if (column_categories[old_col_id] == NC_VAL || column_categories[old_col_id] == NC_STR) {
        // remap
        ignore_vals[old_col_id] = 1;
        mapped_columns.erase({parent_col_id, name});
        columns.erase(old_col_id);
        parent_col.child_columns.erase(name);
        replaced = true;  // to skip duplicate name in column_order
      } else {
        // If this is a nested column but we're trying to insert either (a) a list node into a
        // struct column or (b) a struct node into a list column, we fail
        CUDF_EXPECTS(not((column_categories[old_col_id] == NC_LIST and
                          column_categories[this_col_id] == NC_STRUCT) or
                         (column_categories[old_col_id] == NC_STRUCT and
                          column_categories[this_col_id] == NC_LIST)),
                     "A mix of lists and structs within the same column is not supported");
      }
    }
    CUDF_EXPECTS(parent_col.child_columns.count(name) == 0, "duplicate column name: " + name);
    // move into parent
    device_json_column col(stream, mr);
    initialize_json_columns(this_col_id, col);
    auto inserted = parent_col.child_columns.try_emplace(name, std::move(col)).second;
    CUDF_EXPECTS(inserted, "child column insertion failed, duplicate column name in the parent");
    if (not replaced) parent_col.column_order.push_back(name);
    columns.try_emplace(this_col_id, std::ref(parent_col.child_columns.at(name)));
    mapped_columns.try_emplace(std::make_pair(parent_col_id, name), this_col_id);
  }

  if (is_enabled_mixed_types_as_string) {
    // ignore all children of mixed type columns
    for (auto const this_col_id : unique_col_ids) {
      auto parent_col_id = column_parent_ids[this_col_id];
      if (parent_col_id != parent_node_sentinel and is_mixed_type_column[parent_col_id] == 1) {
        is_mixed_type_column[this_col_id] = 1;
        ignore_vals[this_col_id]          = 1;
        columns.erase(this_col_id);
      }
      // Convert only mixed type columns as string (so to copy), but not its children
      if (parent_col_id != parent_node_sentinel and is_mixed_type_column[parent_col_id] == 0 and
          is_mixed_type_column[this_col_id] == 1)
        column_categories[this_col_id] = NC_STR;
    }
    hipMemcpyAsync(d_column_tree.node_categories.begin(),
                    column_categories.data(),
                    column_categories.size() * sizeof(column_categories[0]),
                    hipMemcpyDefault,
                    stream.value());
  }

  // restore unique_col_ids order
  std::sort(h_range_col_id_it, h_range_col_id_it + num_columns, [](auto const& a, auto const& b) {
    return thrust::get<1>(a) < thrust::get<1>(b);
  });
  // move columns data to device.
  std::vector<json_column_data> columns_data(num_columns);
  for (auto& [col_id, col_ref] : columns) {
    if (col_id == parent_node_sentinel) continue;
    auto& col            = col_ref.get();
    columns_data[col_id] = json_column_data{col.string_offsets.data(),
                                            col.string_lengths.data(),
                                            col.child_offsets.data(),
                                            static_cast<bitmask_type*>(col.validity.data())};
  }

  auto d_ignore_vals = cudf::detail::make_device_uvector_async(
    ignore_vals, stream, rmm::mr::get_current_device_resource());
  auto d_columns_data = cudf::detail::make_device_uvector_async(
    columns_data, stream, rmm::mr::get_current_device_resource());

  // 3. scatter string offsets to respective columns, set validity bits
  thrust::for_each_n(
    rmm::exec_policy(stream),
    thrust::counting_iterator<size_type>(0),
    num_nodes,
    [column_categories = d_column_tree.node_categories.begin(),
     col_ids           = col_ids.begin(),
     row_offsets       = row_offsets.begin(),
     range_begin       = tree.node_range_begin.begin(),
     range_end         = tree.node_range_end.begin(),
     d_ignore_vals     = d_ignore_vals.begin(),
     d_columns_data    = d_columns_data.begin()] __device__(size_type i) {
      if (d_ignore_vals[col_ids[i]]) return;
      auto const node_category = column_categories[col_ids[i]];
      switch (node_category) {
        case NC_STRUCT: set_bit(d_columns_data[col_ids[i]].validity, row_offsets[i]); break;
        case NC_LIST: set_bit(d_columns_data[col_ids[i]].validity, row_offsets[i]); break;
        case NC_STR: [[fallthrough]];
        case NC_VAL:
          if (d_ignore_vals[col_ids[i]]) break;
          set_bit(d_columns_data[col_ids[i]].validity, row_offsets[i]);
          d_columns_data[col_ids[i]].string_offsets[row_offsets[i]] = range_begin[i];
          d_columns_data[col_ids[i]].string_lengths[row_offsets[i]] = range_end[i] - range_begin[i];
          break;
        default: break;
      }
    });

  // 4. scatter List offset
  // copy_if only node's whose parent is list, (node_id, parent_col_id)
  // stable_sort by parent_col_id of {node_id}.
  // For all unique parent_node_id of (i==0, i-1!=i), write start offset.
  //                                  (i==last, i+1!=i), write end offset.
  //    unique_copy_by_key {parent_node_id} {row_offset} to
  //    col[parent_col_id].child_offsets[row_offset[parent_node_id]]

  auto& parent_col_ids = sorted_col_ids;  // reuse sorted_col_ids
  auto parent_col_id   = thrust::make_transform_iterator(
    thrust::make_counting_iterator<size_type>(0),
    cuda::proclaim_return_type<NodeIndexT>(
      [col_ids         = col_ids.begin(),
       parent_node_ids = tree.parent_node_ids.begin()] __device__(size_type node_id) {
        return parent_node_ids[node_id] == parent_node_sentinel ? parent_node_sentinel
                                                                  : col_ids[parent_node_ids[node_id]];
      }));
  auto const list_children_end = thrust::copy_if(
    rmm::exec_policy(stream),
    thrust::make_zip_iterator(thrust::make_counting_iterator<size_type>(0), parent_col_id),
    thrust::make_zip_iterator(thrust::make_counting_iterator<size_type>(0), parent_col_id) +
      num_nodes,
    thrust::make_counting_iterator<size_type>(0),
    thrust::make_zip_iterator(node_ids.begin(), parent_col_ids.begin()),
    [d_ignore_vals     = d_ignore_vals.begin(),
     parent_node_ids   = tree.parent_node_ids.begin(),
     column_categories = d_column_tree.node_categories.begin(),
     col_ids           = col_ids.begin()] __device__(size_type node_id) {
      auto parent_node_id = parent_node_ids[node_id];
      return parent_node_id != parent_node_sentinel and
             column_categories[col_ids[parent_node_id]] == NC_LIST and
             (!d_ignore_vals[col_ids[parent_node_id]]);
    });

  auto const num_list_children =
    list_children_end - thrust::make_zip_iterator(node_ids.begin(), parent_col_ids.begin());
  thrust::stable_sort_by_key(rmm::exec_policy(stream),
                             parent_col_ids.begin(),
                             parent_col_ids.begin() + num_list_children,
                             node_ids.begin());
  thrust::for_each_n(
    rmm::exec_policy(stream),
    thrust::make_counting_iterator<size_type>(0),
    num_list_children,
    [node_ids        = node_ids.begin(),
     parent_node_ids = tree.parent_node_ids.begin(),
     parent_col_ids  = parent_col_ids.begin(),
     row_offsets     = row_offsets.begin(),
     d_columns_data  = d_columns_data.begin(),
     num_list_children] __device__(size_type i) {
      auto const node_id        = node_ids[i];
      auto const parent_node_id = parent_node_ids[node_id];
      // scatter to list_offset
      if (i == 0 or parent_node_ids[node_ids[i - 1]] != parent_node_id) {
        d_columns_data[parent_col_ids[i]].child_offsets[row_offsets[parent_node_id]] =
          row_offsets[node_id];
      }
      // last value of list child_offset is its size.
      if (i == num_list_children - 1 or parent_node_ids[node_ids[i + 1]] != parent_node_id) {
        d_columns_data[parent_col_ids[i]].child_offsets[row_offsets[parent_node_id] + 1] =
          row_offsets[node_id] + 1;
      }
    });

  // 5. scan on offsets.
  for (auto& [id, col_ref] : columns) {
    auto& col = col_ref.get();
    if (col.type == json_col_t::StringColumn) {
      thrust::inclusive_scan(rmm::exec_policy(stream),
                             col.string_offsets.begin(),
                             col.string_offsets.end(),
                             col.string_offsets.begin(),
                             thrust::maximum<json_column::row_offset_t>{});
    } else if (col.type == json_col_t::ListColumn) {
      thrust::inclusive_scan(rmm::exec_policy(stream),
                             col.child_offsets.begin(),
                             col.child_offsets.end(),
                             col.child_offsets.begin(),
                             thrust::maximum<json_column::row_offset_t>{});
    }
  }
}

std::pair<std::unique_ptr<column>, std::vector<column_name_info>> device_json_column_to_cudf_column(
  device_json_column& json_col,
  device_span<SymbolT const> d_input,
  cudf::io::parse_options const& options,
  std::optional<schema_element> schema,
  rmm::cuda_stream_view stream,
  rmm::mr::device_memory_resource* mr)
{
  CUDF_FUNC_RANGE();
  auto validity_size_check = [](device_json_column& json_col) {
    CUDF_EXPECTS(json_col.validity.size() >= bitmask_allocation_size_bytes(json_col.num_rows),
                 "valid_count is too small");
  };
  auto make_validity = [stream, validity_size_check](
                         device_json_column& json_col) -> std::pair<rmm::device_buffer, size_type> {
    validity_size_check(json_col);
    auto null_count = cudf::detail::null_count(
      static_cast<bitmask_type*>(json_col.validity.data()), 0, json_col.num_rows, stream);
    // full null_mask is always required for parse_data
    return {std::move(json_col.validity), null_count};
    // Note: json_col modified here, moves this memory
  };

  auto get_child_schema = [schema](auto child_name) -> std::optional<schema_element> {
    if (schema.has_value()) {
      auto const result = schema.value().child_types.find(child_name);
      if (result != std::end(schema.value().child_types)) { return result->second; }
    }
    return {};
  };

  switch (json_col.type) {
    case json_col_t::StringColumn: {
      // move string_offsets to GPU and transform to string column
      auto const col_size      = json_col.string_offsets.size();
      using char_length_pair_t = thrust::pair<char const*, size_type>;
      CUDF_EXPECTS(json_col.string_offsets.size() == json_col.string_lengths.size(),
                   "string offset, string length mismatch");
      rmm::device_uvector<char_length_pair_t> d_string_data(col_size, stream);
      // TODO how about directly storing pair<char*, size_t> in json_column?
      auto offset_length_it =
        thrust::make_zip_iterator(json_col.string_offsets.begin(), json_col.string_lengths.begin());

      data_type target_type{};

      if (schema.has_value()) {
#ifdef NJP_DEBUG_PRINT
        std::cout << "-> explicit type: "
                  << (schema.has_value() ? std::to_string(static_cast<int>(schema->type.id()))
                                         : "n/a");
#endif
        target_type = schema.value().type;
      } else if (json_col.forced_as_string_column) {
        target_type = data_type{type_id::STRING};
      }
      // Infer column type, if we don't have an explicit type for it
      else {
        target_type = cudf::io::detail::infer_data_type(
          options.json_view(), d_input, offset_length_it, col_size, stream);
      }

      auto [result_bitmask, null_count] = make_validity(json_col);
      // Convert strings to the inferred data type
      auto col = parse_data(d_input.data(),
                            offset_length_it,
                            col_size,
                            target_type,
                            std::move(result_bitmask),
                            null_count,
                            options.view(),
                            stream,
                            mr);

      // Reset nullable if we do not have nulls
      // This is to match the existing JSON reader's behaviour:
      // - Non-string columns will always be returned as nullable
      // - String columns will be returned as nullable, iff there's at least one null entry
      if (col->null_count() == 0) { col->set_null_mask(rmm::device_buffer{0, stream, mr}, 0); }

      // For string columns return ["offsets", "char"] schema
      if (target_type.id() == type_id::STRING) {
        return {std::move(col), std::vector<column_name_info>{{"offsets"}, {"chars"}}};
      }
      // Non-string leaf-columns (e.g., numeric) do not have child columns in the schema
      return {std::move(col), std::vector<column_name_info>{}};
    }
    case json_col_t::StructColumn: {
      std::vector<std::unique_ptr<column>> child_columns;
      std::vector<column_name_info> column_names{};
      size_type num_rows{json_col.num_rows};
      // Create children columns
      for (auto const& col_name : json_col.column_order) {
        auto const& col = json_col.child_columns.find(col_name);
        column_names.emplace_back(col->first);
        auto& child_col            = col->second;
        auto [child_column, names] = device_json_column_to_cudf_column(
          child_col, d_input, options, get_child_schema(col_name), stream, mr);
        CUDF_EXPECTS(num_rows == child_column->size(),
                     "All children columns must have the same size");
        child_columns.push_back(std::move(child_column));
        column_names.back().children = names;
      }
      auto [result_bitmask, null_count] = make_validity(json_col);
      // The null_mask is set after creation of struct column is to skip the superimpose_nulls and
      // null validation applied in make_structs_column factory, which is not needed for json
      auto ret_col = make_structs_column(num_rows, std::move(child_columns), 0, {}, stream, mr);
      if (null_count != 0) { ret_col->set_null_mask(std::move(result_bitmask), null_count); }
      return {std::move(ret_col), column_names};
    }
    case json_col_t::ListColumn: {
      size_type num_rows = json_col.child_offsets.size() - 1;
      std::vector<column_name_info> column_names{};
      column_names.emplace_back("offsets");
      column_names.emplace_back(
        json_col.child_columns.empty() ? list_child_name : json_col.child_columns.begin()->first);

      // Note: json_col modified here, reuse the memory
      auto offsets_column = std::make_unique<column>(data_type{type_id::INT32},
                                                     num_rows + 1,
                                                     json_col.child_offsets.release(),
                                                     rmm::device_buffer{},
                                                     0);
      // Create children column
      auto [child_column, names] =
        json_col.child_columns.empty()
          ? std::pair<std::unique_ptr<column>,
                      // EMPTY type could not used because gather throws exception on EMPTY type.
                      std::vector<column_name_info>>{std::make_unique<column>(
                                                       data_type{type_id::INT8},
                                                       0,
                                                       rmm::device_buffer{},
                                                       rmm::device_buffer{},
                                                       0),
                                                     std::vector<column_name_info>{}}
          : device_json_column_to_cudf_column(
              json_col.child_columns.begin()->second,
              d_input,
              options,
              get_child_schema(json_col.child_columns.begin()->first),
              stream,
              mr);
      column_names.back().children      = names;
      auto [result_bitmask, null_count] = make_validity(json_col);
      auto ret_col                      = make_lists_column(num_rows,
                                       std::move(offsets_column),
                                       std::move(child_column),
                                       0,
                                       rmm::device_buffer{0, stream, mr},
                                       stream,
                                       mr);
      // The null_mask is set after creation of list column is to skip the purge_nonempty_nulls and
      // null validation applied in make_lists_column factory, which is not needed for json
      // parent column cannot be null when its children is non-empty in JSON
      if (null_count != 0) { ret_col->set_null_mask(std::move(result_bitmask), null_count); }
      return {std::move(ret_col), std::move(column_names)};
    }
    default: CUDF_FAIL("Unsupported column type"); break;
  }
}

table_with_metadata device_parse_nested_json(device_span<SymbolT const> d_input,
                                             cudf::io::json_reader_options const& options,
                                             rmm::cuda_stream_view stream,
                                             rmm::mr::device_memory_resource* mr)
{
  CUDF_FUNC_RANGE();

  auto gpu_tree = [&]() {
    // Parse the JSON and get the token stream
    const auto [tokens_gpu, token_indices_gpu] =
      get_token_stream(d_input, options, stream, rmm::mr::get_current_device_resource());
    // gpu tree generation
    return get_tree_representation(tokens_gpu,
                                   token_indices_gpu,
                                   options.is_enabled_mixed_types_as_string(),
                                   stream,
                                   rmm::mr::get_current_device_resource());
  }();  // IILE used to free memory of token data.
#ifdef NJP_DEBUG_PRINT
  auto h_input = cudf::detail::make_host_vector_async(d_input, stream);
  print_tree(h_input, gpu_tree, stream);
#endif

  bool const is_array_of_arrays = [&]() {
    std::array<node_t, 2> h_node_categories = {NC_ERR, NC_ERR};
    auto const size_to_copy                 = std::min(size_t{2}, gpu_tree.node_categories.size());
    CUDF_CUDA_TRY(hipMemcpyAsync(h_node_categories.data(),
                                  gpu_tree.node_categories.data(),
                                  sizeof(node_t) * size_to_copy,
                                  hipMemcpyDefault,
                                  stream.value()));
    stream.synchronize();
    if (options.is_enabled_lines()) return h_node_categories[0] == NC_LIST;
    return h_node_categories[0] == NC_LIST and h_node_categories[1] == NC_LIST;
  }();

  auto [gpu_col_id, gpu_row_offsets] =
    records_orient_tree_traversal(d_input,
                                  gpu_tree,
                                  is_array_of_arrays,
                                  options.is_enabled_lines(),
                                  stream,
                                  rmm::mr::get_current_device_resource());

  device_json_column root_column(stream, mr);
  root_column.type = json_col_t::ListColumn;
  root_column.child_offsets.resize(2, stream);
  thrust::fill(rmm::exec_policy(stream),
               root_column.child_offsets.begin(),
               root_column.child_offsets.end(),
               0);

  // Get internal JSON column
  make_device_json_column(d_input,
                          gpu_tree,
                          gpu_col_id,
                          gpu_row_offsets,
                          root_column,
                          is_array_of_arrays,
                          options,
                          stream,
                          mr);

  // data_root refers to the root column of the data represented by the given JSON string
  auto& data_root =
    options.is_enabled_lines() ? root_column : root_column.child_columns.begin()->second;

  // Zero row entries
  if (data_root.type == json_col_t::ListColumn && data_root.child_columns.empty()) {
    return table_with_metadata{std::make_unique<table>(std::vector<std::unique_ptr<column>>{})};
  }

  // Verify that we were in fact given a list of structs (or in JSON speech: an array of objects)
  auto constexpr single_child_col_count = 1;
  CUDF_EXPECTS(data_root.type == json_col_t::ListColumn and
                 data_root.child_columns.size() == single_child_col_count and
                 data_root.child_columns.begin()->second.type ==
                   (is_array_of_arrays ? json_col_t::ListColumn : json_col_t::StructColumn),
               "Input needs to be an array of arrays or an array of (nested) objects");

  // Slice off the root list column, which has only a single row that contains all the structs
  auto& root_struct_col = data_root.child_columns.begin()->second;

  // Initialize meta data to be populated while recursing through the tree of columns
  std::vector<std::unique_ptr<column>> out_columns;
  std::vector<column_name_info> out_column_names;
  auto parse_opt = parsing_options(options, stream);

  // Iterate over the struct's child columns and convert to cudf column
  size_type column_index = 0;
  for (auto const& col_name : root_struct_col.column_order) {
    auto& json_col = root_struct_col.child_columns.find(col_name)->second;
    // Insert this columns name into the schema
    out_column_names.emplace_back(col_name);

    std::optional<schema_element> child_schema_element = std::visit(
      cudf::detail::visitor_overload{
        [column_index](std::vector<data_type> const& user_dtypes) -> std::optional<schema_element> {
          return (static_cast<std::size_t>(column_index) < user_dtypes.size())
                   ? std::optional<schema_element>{{user_dtypes[column_index]}}
                   : std::optional<schema_element>{};
        },
        [col_name](
          std::map<std::string, data_type> const& user_dtypes) -> std::optional<schema_element> {
          return (user_dtypes.find(col_name) != std::end(user_dtypes))
                   ? std::optional<schema_element>{{user_dtypes.find(col_name)->second}}
                   : std::optional<schema_element>{};
        },
        [col_name](std::map<std::string, schema_element> const& user_dtypes)
          -> std::optional<schema_element> {
          return (user_dtypes.find(col_name) != std::end(user_dtypes))
                   ? user_dtypes.find(col_name)->second
                   : std::optional<schema_element>{};
        }},
      options.get_dtypes());
#ifdef NJP_DEBUG_PRINT
    auto debug_schema_print = [](auto ret) {
      std::cout << ", type id: "
                << (ret.has_value() ? std::to_string(static_cast<int>(ret->type.id())) : "n/a")
                << ", with " << (ret.has_value() ? ret->child_types.size() : 0) << " children"
                << "\n";
    };
    std::visit(
      cudf::detail::visitor_overload{[column_index](std::vector<data_type> const&) {
                                       std::cout << "Column by index: #" << column_index;
                                     },
                                     [col_name](std::map<std::string, data_type> const&) {
                                       std::cout << "Column by flat name: '" << col_name;
                                     },
                                     [col_name](std::map<std::string, schema_element> const&) {
                                       std::cout << "Column by nested name: #" << col_name;
                                     }},
      options.get_dtypes());
    debug_schema_print(child_schema_element);
#endif

    // Get this JSON column's cudf column and schema info, (modifies json_col)
    auto [cudf_col, col_name_info] = device_json_column_to_cudf_column(
      json_col, d_input, parse_opt, child_schema_element, stream, mr);
    // TODO: RangeIndex as DataFrame.columns names for array of arrays
    // if (is_array_of_arrays) {
    //   col_name_info.back().name = "";
    // }

    out_column_names.back().children = std::move(col_name_info);
    out_columns.emplace_back(std::move(cudf_col));

    column_index++;
  }

  return table_with_metadata{std::make_unique<table>(std::move(out_columns)), {out_column_names}};
}

}  // namespace cudf::io::json::detail
