#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2020, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "json_common.h"
#include "json_gpu.h"

#include <io/csv/datetime.cuh>
#include <io/utilities/parsing_utils.cuh>

#include <cudf/detail/utilities/hash_functions.cuh>
#include <cudf/detail/utilities/trie.cuh>
#include <cudf/fixed_point/fixed_point.hpp>
#include <cudf/lists/list_view.cuh>
#include <cudf/strings/string_view.cuh>
#include <cudf/utilities/bit.hpp>
#include <cudf/utilities/span.hpp>
#include <cudf/utilities/traits.hpp>
#include <cudf/utilities/type_dispatcher.hpp>

#include <rmm/cuda_stream_view.hpp>
#include <rmm/device_buffer.hpp>
#include <rmm/device_vector.hpp>
#include <rmm/exec_policy.hpp>

#include <thrust/detail/copy.h>
#include <thrust/find.h>

using cudf::detail::device_span;

namespace cudf {
namespace io {
namespace json {
namespace gpu {
using namespace ::cudf;

using string_pair = std::pair<const char *, size_t>;

namespace {
/**
 * @brief CUDA Kernel that adjusts the row range to exclude the character outside of the top level
 * brackets.
 *
 * The top level brackets characters are excluded from the resulting range.
 *
 * @param[in] begin Pointer to the first character in the row
 * @param[in] end pointer to the first character after the row
 */
__device__ std::pair<char const *, char const *> limit_range_to_brackets(char const *begin,
                                                                         char const *end)
{
  begin = thrust::find_if(
    thrust::seq, begin, end, [] __device__(auto c) { return c == '[' || c == '{'; });
  end = thrust::find_if(thrust::seq,
                        thrust::make_reverse_iterator(end),
                        thrust::make_reverse_iterator(++begin),
                        [](auto c) { return c == ']' || c == '}'; })
          .base();
  return {begin, --end};
}

/**
 * @brief Find the first JSON object key in the range.
 *
 * Assumes that begin is not in the middle of a field.
 *
 * @param[in] begin Pointer to the first character in the parsing range
 * @param[in] end pointer to the first character after the parsing range
 * @param[in] quotechar The character used to denote quotes
 *
 * @return Begin and end iterators of the key name; (`end`, `end`) if a key is not found
 */
__device__ std::pair<char const *, char const *> get_next_key(char const *begin,
                                                              char const *end,
                                                              char quotechar)
{
  // Key starts after the first quote
  auto const key_begin = thrust::find(thrust::seq, begin, end, quotechar) + 1;
  if (key_begin > end) return {end, end};

  // Key ends after the next unescaped quote
  auto const key_end_pair = thrust::mismatch(
    thrust::seq, key_begin, end - 1, key_begin + 1, [quotechar] __device__(auto prev_ch, auto ch) {
      return !(ch == quotechar && prev_ch != '\\');
    });

  return {key_begin, key_end_pair.second};
}

/**
 * @brief Decodes a numeric value base on templated cudf type T with specified
 * base.
 *
 * @param[in] begin Beginning of the character string
 * @param[in] end End of the character string
 * @param opts The global parsing behavior options
 *
 * @return The parsed numeric value
 */
template <typename T, int base>
__inline__ __device__ T decode_value(const char *begin,
                                     uint64_t end,
                                     parse_options_view const &opts)
{
  return cudf::io::gpu::parse_numeric<T, base>(begin, end, opts);
}

/**
 * @brief Decodes a numeric value base on templated cudf type T
 *
 * @param[in] begin Beginning of the character string
 * @param[in] end End of the character string
 * @param opts The global parsing behavior options
 *
 * @return The parsed numeric value
 */
template <typename T>
__inline__ __device__ T decode_value(const char *begin,
                                     const char *end,
                                     parse_options_view const &opts)
{
  return cudf::io::gpu::parse_numeric<T>(begin, end, opts);
}

/**
 * @brief Decodes a timestamp_D
 *
 * @param[in] begin Beginning of the character string
 * @param[in] end End of the character string
 * @param opts The global parsing behavior options
 *
 * @return The parsed timestamp_D
 */
template <>
__inline__ __device__ cudf::timestamp_D decode_value(const char *begin,
                                                     const char *end,
                                                     parse_options_view const &opts)
{
  return cudf::timestamp_D{cudf::duration_D{parseDateFormat(begin, end, opts.dayfirst)}};
}

/**
 * @brief Decodes a timestamp_s
 *
 * @param[in] begin Beginning of the character string
 * @param[in] end End of the character string
 * @param opts The global parsing behavior options
 *
 * @return The parsed timestamp_s
 */
template <>
__inline__ __device__ cudf::timestamp_s decode_value(const char *begin,
                                                     const char *end,
                                                     parse_options_view const &opts)
{
  auto milli = parseDateTimeFormat(begin, end, opts.dayfirst);
  return cudf::timestamp_s{cudf::duration_s{milli / 1000}};
}

/**
 * @brief Decodes a timestamp_ms
 *
 * @param[in] begin Beginning of the character string
 * @param[in] end End of the character string
 * @param opts The global parsing behavior options
 *
 * @return The parsed timestamp_ms
 */
template <>
__inline__ __device__ cudf::timestamp_ms decode_value(const char *begin,
                                                      const char *end,
                                                      parse_options_view const &opts)
{
  auto milli = parseDateTimeFormat(begin, end, opts.dayfirst);
  return cudf::timestamp_ms{cudf::duration_ms{milli}};
}

/**
 * @brief Decodes a timestamp_us
 *
 * @param[in] begin Beginning of the character string
 * @param[in] end End of the character string
 * @param opts The global parsing behavior options
 *
 * @return The parsed timestamp_us
 */
template <>
__inline__ __device__ cudf::timestamp_us decode_value(const char *begin,
                                                      const char *end,
                                                      parse_options_view const &opts)
{
  auto milli = parseDateTimeFormat(begin, end, opts.dayfirst);
  return cudf::timestamp_us{cudf::duration_us{milli * 1000}};
}

/**
 * @brief Decodes a timestamp_ns
 *
 * @param[in] begin Beginning of the character string
 * @param[in] end End of the character string
 * @param opts The global parsing behavior options
 *
 * @return The parsed timestamp_ns
 */
template <>
__inline__ __device__ cudf::timestamp_ns decode_value(const char *begin,
                                                      const char *end,
                                                      parse_options_view const &opts)
{
  auto milli = parseDateTimeFormat(begin, end, opts.dayfirst);
  return cudf::timestamp_ns{cudf::duration_ns{milli * 1000000}};
}

#ifndef DURATION_DECODE_VALUE
#define DURATION_DECODE_VALUE(Type)                                     \
  template <>                                                           \
  __inline__ __device__ Type decode_value(                              \
    const char *begin, const char *end, parse_options_view const &opts) \
  {                                                                     \
    return Type{parseTimeDeltaFormat<Type>(begin, 0, end - begin)};     \
  }
#endif
DURATION_DECODE_VALUE(duration_D)
DURATION_DECODE_VALUE(duration_s)
DURATION_DECODE_VALUE(duration_ms)
DURATION_DECODE_VALUE(duration_us)
DURATION_DECODE_VALUE(duration_ns)

// The purpose of these is merely to allow compilation ONLY
template <>
__inline__ __device__ cudf::string_view decode_value(const char *begin,
                                                     const char *end,
                                                     parse_options_view const &opts)
{
  return cudf::string_view{};
}

template <>
__inline__ __device__ cudf::dictionary32 decode_value(const char *begin,
                                                      const char *end,
                                                      parse_options_view const &opts)
{
  return cudf::dictionary32{};
}

template <>
__inline__ __device__ cudf::list_view decode_value(const char *begin,
                                                   const char *end,
                                                   parse_options_view const &opts)
{
  return cudf::list_view{};
}
template <>
__inline__ __device__ cudf::struct_view decode_value(const char *begin,
                                                     const char *end,
                                                     parse_options_view const &opts)
{
  return cudf::struct_view{};
}

template <>
__inline__ __device__ numeric::decimal32 decode_value(const char *begin,
                                                      const char *end,
                                                      parse_options_view const &opts)
{
  return numeric::decimal32{};
}

template <>
__inline__ __device__ numeric::decimal64 decode_value(const char *begin,
                                                      const char *end,
                                                      parse_options_view const &opts)
{
  return numeric::decimal64{};
}

/**
 * @brief Functor for converting plain text data to cuDF data type value.
 */
struct ConvertFunctor {
  /**
   * @brief Template specialization for operator() for types whose values can be
   * convertible to a 0 or 1 to represent false/true. The converting is done by
   * checking against the default and user-specified true/false values list.
   *
   * It is handled here rather than within convertStrToValue() as that function
   * is used by other types (ex. timestamp) that aren't 'booleable'.
   */
  template <typename T, typename std::enable_if_t<std::is_integral<T>::value> * = nullptr>
  __host__ __device__ __forceinline__ bool operator()(char const *begin,
                                                      char const *end,
                                                      void *output_column,
                                                      cudf::size_type row,
                                                      const parse_options_view &opts)
  {
    T &value{static_cast<T *>(output_column)[row]};

    // Check for user-specified true/false values first, where the output is
    // replaced with 1/0 respectively
    value = [&opts, end, begin]() -> T {
      if (serialized_trie_contains(opts.trie_true, begin, end - begin)) {
        return 1;
      } else if (serialized_trie_contains(opts.trie_false, begin, end - begin)) {
        return 0;
      } else {
        return decode_value<T>(begin, end - 1, opts);
      }
    }();

    return true;
  }

  /**
   * @brief Dispatch for floating points, which are set to NaN if the input
   * is not valid. In such case, the validity mask is set to zero too.
   */
  template <typename T, typename std::enable_if_t<std::is_floating_point<T>::value> * = nullptr>
  __host__ __device__ __forceinline__ bool operator()(char const *begin,
                                                      char const *end,
                                                      void *out_buffer,
                                                      size_t row,
                                                      parse_options_view const &opts)
  {
    auto &value{static_cast<T *>(out_buffer)[row]};
    value = decode_value<T>(begin, end - 1, opts);
    return !std::isnan(value);
  }

  /**
   * @brief Default template operator() dispatch specialization all data types
   * (including wrapper types) that is not covered by above.
   */
  template <typename T,
            typename std::enable_if_t<!std::is_floating_point<T>::value and
                                      !std::is_integral<T>::value> * = nullptr>
  __host__ __device__ __forceinline__ bool operator()(char const *begin,
                                                      char const *end,
                                                      void *output_column,
                                                      cudf::size_type row,
                                                      const parse_options_view &opts)
  {
    T &value{static_cast<T *>(output_column)[row]};
    value = decode_value<T>(begin, end - 1, opts);

    return true;
  }
};

/**
 * @brief Checks whether the given character is a whitespace character.
 *
 * @param[in] ch The character to check
 *
 * @return True if the input is whitespace, False otherwise
 */
__inline__ __device__ bool is_whitespace(char ch) { return ch == '\t' || ch == ' '; }

/**
 * @brief Adjusts the range to ignore starting/trailing whitespace and quotation characters.
 *
 * @param[in] begin Pointer to the first character in the parsing range
 * @param[in] end pointer to the first character after the parsing range
 * @param[in] quotechar The character used to denote quotes; '\0' if none
 *
 * @return Trimmed range
 */
__inline__ __device__ std::pair<char const *, char const *> trim_whitespaces_quotes(
  char const *begin, char const *end, char quotechar = '\0')
{
  auto not_whitespace = [] __device__(auto c) { return !is_whitespace(c); };

  begin = thrust::find_if(thrust::seq, begin, end, not_whitespace);
  end   = thrust::find_if(thrust::seq,
                        thrust::make_reverse_iterator(end),
                        thrust::make_reverse_iterator(begin),
                        not_whitespace)
          .base();

  return {(*begin == quotechar) ? ++begin : begin, (*(end - 1) == quotechar) ? end - 1 : end};
}

/**
 * @brief Returns true is the input character is a valid digit.
 * Supports both decimal and hexadecimal digits (uppercase and lowercase).
 *
 * @param c Character to check
 * @param is_hex Whether to check as a hexadecimal
 *
 * @return `true` if it is digit-like, `false` otherwise
 */
__device__ __inline__ bool is_digit(char c, bool is_hex = false)
{
  if (c >= '0' && c <= '9') return true;

  if (is_hex) {
    if (c >= 'A' && c <= 'F') return true;
    if (c >= 'a' && c <= 'f') return true;
  }

  return false;
}

/**
 * @brief Returns true if the counters indicate a potentially valid float.
 * False positives are possible because positions are not taken into account.
 * For example, field "e.123-" would match the pattern.
 */
__device__ __inline__ bool is_like_float(
  long len, long digit_cnt, long decimal_cnt, long dash_cnt, long exponent_cnt)
{
  // Can't have more than one exponent and one decimal point
  if (decimal_cnt > 1) return false;
  if (exponent_cnt > 1) return false;
  // Without the exponent or a decimal point, this is an integer, not a float
  if (decimal_cnt == 0 && exponent_cnt == 0) return false;

  // Can only have one '-' per component
  if (dash_cnt > 1 + exponent_cnt) return false;

  // If anything other than these characters is present, it's not a float
  if (digit_cnt + decimal_cnt + dash_cnt + exponent_cnt != len) return false;

  // Needs at least 1 digit, 2 if exponent is present
  if (digit_cnt < 1 + exponent_cnt) return false;

  return true;
}

/**
 * @brief Contains information on a JSON file field.
 */
struct field_descriptor {
  cudf::size_type column;
  char const *value_begin;
  char const *value_end;
};

/**
 * @brief Parse the first field in the given range and return its descriptor.
 *
 * @param[in] begin Pointer to the first character in the parsing range
 * @param[in] end pointer to the first character after the parsing range
 * @param[in] opts The global parsing behavior options
 * @param[in] field_idx Index of the current field in the input row
 * @param[in] col_map Pointer to the (column name hash -> solumn index) map in device memory.
 * nullptr is passed when the input file does not consist of objects.
 * @return Descriptor of the parsed field
 */
__device__ field_descriptor next_field_descriptor(const char *begin,
                                                  const char *end,
                                                  parse_options_view const &opts,
                                                  cudf::size_type field_idx,
                                                  col_map_type *col_map)
{
  auto const desc_pre_trim =
    col_map == nullptr
      // No key - column and begin are trivial
      ? field_descriptor{field_idx, begin, cudf::io::gpu::seek_field_end(begin, end, opts, true)}
      : [&]() {
          auto const key_range = get_next_key(begin, end, opts.quotechar);
          auto const key_hash  = MurmurHash3_32<cudf::string_view>{}(
            cudf::string_view(key_range.first, key_range.second - key_range.first));
          auto const hash_col = col_map->find(key_hash);
          // Fall back to field index if not found (parsing error)
          auto const column = (hash_col != col_map->end()) ? (*hash_col).second : field_idx;

          // Skip the colon between the key and the value
          auto const value_begin = thrust::find(thrust::seq, key_range.second, end, ':') + 1;
          return field_descriptor{
            column, value_begin, cudf::io::gpu::seek_field_end(value_begin, end, opts, true)};
        }();

  // Modify start & end to ignore whitespace and quotechars
  auto const trimmed_value_range =
    trim_whitespaces_quotes(desc_pre_trim.value_begin, desc_pre_trim.value_end, opts.quotechar);
  return {desc_pre_trim.column, trimmed_value_range.first, trimmed_value_range.second};
}

/**
 * @brief Returns the range that contains the data in a given row.
 *
 * Excludes the top-level brackets.
 *
 * @param[in] data Pointer to the JSON data in device memory
 * @param[in] data_size Size of the data buffer, in bytes
 * @param[in] rec_starts The offset of each row in the input
 * @param[in] num_rows The number of lines/rows
 * @param[in] row Index of the row for which the range is returned
 *
 * @return The begin and end iterators of the row data.
 */
__device__ std::pair<char const *, char const *> get_row_data_range(
  device_span<char const> const data, device_span<uint64_t const> const row_offsets, size_type row)
{
  auto const row_begin = data.begin() + row_offsets[row];
  auto const row_end =
    data.begin() + ((row < row_offsets.size() - 1) ? row_offsets[row + 1] : data.size());
  return limit_range_to_brackets(row_begin, row_end);
}

/**
 * @brief CUDA kernel that parses and converts plain text data into cuDF column data.
 *
 * Data is processed one record at a time
 *
 * @param[in] opts A set of parsing options
 * @param[in] data The entire data to read
 * @param[in] row_offsets The offset of each row in the input
 * @param[in] column_types The data type of each column
 * @param[in] col_map Pointer to the (column name hash -> solumn index) map in device memory.
 * nullptr is passed when the input file does not consist of objects.
 * @param[out] output_columns The output column data
 * @param[out] valid_fields The bitmaps indicating whether column fields are valid
 * @param[out] num_valid_fields The numbers of valid fields in columns
 */
__global__ void convert_data_to_columns_kernel(parse_options_view opts,
                                               device_span<char const> const data,
                                               device_span<uint64_t const> const row_offsets,
                                               device_span<data_type const> const column_types,
                                               col_map_type *col_map,
                                               device_span<void *const> const output_columns,
                                               device_span<bitmask_type *const> const valid_fields,
                                               device_span<cudf::size_type> const num_valid_fields)
{
  const auto rec_id = threadIdx.x + (blockDim.x * blockIdx.x);
  if (rec_id >= row_offsets.size()) return;

  auto const row_data_range = get_row_data_range(data, row_offsets, rec_id);

  auto current = row_data_range.first;
  for (size_type input_field_index = 0;
       input_field_index < column_types.size() && current < row_data_range.second;
       input_field_index++) {
    auto const desc =
      next_field_descriptor(current, row_data_range.second, opts, input_field_index, col_map);
    auto const value_len = static_cast<size_t>(std::max(desc.value_end - desc.value_begin, 0L));

    current = desc.value_end + 1;

    // Empty fields are not legal values
    if (!serialized_trie_contains(opts.trie_na, desc.value_begin, value_len)) {
      // Type dispatcher does not handle strings
      if (column_types[desc.column].id() == type_id::STRING) {
        auto str_list           = static_cast<string_pair *>(output_columns[desc.column]);
        str_list[rec_id].first  = desc.value_begin;
        str_list[rec_id].second = value_len;

        // set the valid bitmap - all bits were set to 0 to start
        set_bit(valid_fields[desc.column], rec_id);
        atomicAdd(&num_valid_fields[desc.column], 1);
      } else {
        if (cudf::type_dispatcher(column_types[desc.column],
                                  ConvertFunctor{},
                                  desc.value_begin,
                                  desc.value_end,
                                  output_columns[desc.column],
                                  rec_id,
                                  opts)) {
          // set the valid bitmap - all bits were set to 0 to start
          set_bit(valid_fields[desc.column], rec_id);
          atomicAdd(&num_valid_fields[desc.column], 1);
        }
      }
    } else if (column_types[desc.column].id() == type_id::STRING) {
      auto str_list           = static_cast<string_pair *>(output_columns[desc.column]);
      str_list[rec_id].first  = nullptr;
      str_list[rec_id].second = 0;
    }
  }
}

/**
 * @brief CUDA kernel that processes a buffer of data and determines information about the
 * column types within.
 *
 * Data is processed in one row/record at a time, so the number of total
 * threads (tid) is equal to the number of rows.
 *
 * @param[in] opts A set of parsing options
 * @param[in] data Input data buffer
 * @param[in] rec_starts The offset of each row in the input
 * @param[in] col_map Pointer to the (column name hash -> column index) map in device memory.
 * nullptr is passed when the input file does not consist of objects.
 * @param[in] num_columns The number of columns of input data
 * @param[out] column_infos The count for each column data type
 */
__global__ void detect_data_types_kernel(
  parse_options_view const opts,
  device_span<char const> const data,
  device_span<uint64_t const> const row_offsets,
  col_map_type *col_map,
  int num_columns,
  device_span<cudf::io::column_type_histogram> const column_infos)
{
  auto const rec_id = threadIdx.x + (blockDim.x * blockIdx.x);
  if (rec_id >= row_offsets.size()) return;

  auto const are_rows_objects = col_map != nullptr;
  auto const row_data_range   = get_row_data_range(data, row_offsets, rec_id);

  size_type input_field_index = 0;
  for (auto current = row_data_range.first;
       input_field_index < num_columns && current < row_data_range.second;
       input_field_index++) {
    auto const desc =
      next_field_descriptor(current, row_data_range.second, opts, input_field_index, col_map);
    auto const value_len = static_cast<size_t>(std::max(desc.value_end - desc.value_begin, 0L));

    // Advance to the next field; +1 to skip the delimiter
    current = desc.value_end + 1;

    // Checking if the field is empty/valid
    if (serialized_trie_contains(opts.trie_na, desc.value_begin, value_len)) {
      // Increase the null count for array rows, where the null count is initialized to zero.
      if (!are_rows_objects) { atomicAdd(&column_infos[desc.column].null_count, 1); }
      continue;
    } else if (are_rows_objects) {
      // For files with object rows, null count is initialized to row count. The value is decreased
      // here for every valid field.
      atomicAdd(&column_infos[desc.column].null_count, -1);
    }
    // Don't need counts to detect strings, any field in quotes is deduced to be a string
    if (*(desc.value_begin - 1) == opts.quotechar && *desc.value_end == opts.quotechar) {
      atomicAdd(&column_infos[desc.column].string_count, 1);
      continue;
    }

    int digit_count    = 0;
    int decimal_count  = 0;
    int slash_count    = 0;
    int dash_count     = 0;
    int plus_count     = 0;
    int colon_count    = 0;
    int exponent_count = 0;
    int other_count    = 0;

    const bool maybe_hex =
      ((value_len > 2 && *desc.value_begin == '0' && *(desc.value_begin + 1) == 'x') ||
       (value_len > 3 && *desc.value_begin == '-' && *(desc.value_begin + 1) == '0' &&
        *(desc.value_begin + 2) == 'x'));
    for (auto pos = desc.value_begin; pos < desc.value_end; ++pos) {
      if (is_digit(*pos, maybe_hex)) {
        digit_count++;
        continue;
      }
      // Looking for unique characters that will help identify column types
      switch (*pos) {
        case '.': decimal_count++; break;
        case '-': dash_count++; break;
        case '+': plus_count++; break;
        case '/': slash_count++; break;
        case ':': colon_count++; break;
        case 'e':
        case 'E':
          if (!maybe_hex && pos > desc.value_begin && pos < desc.value_end - 1) exponent_count++;
          break;
        default: other_count++; break;
      }
    }

    // Integers have to have the length of the string
    int int_req_number_cnt = value_len;
    // Off by one if they start with a minus sign
    if ((*desc.value_begin == '-' || *desc.value_begin == '+') && value_len > 1) {
      --int_req_number_cnt;
    }
    // Off by one if they are a hexadecimal number
    if (maybe_hex) { --int_req_number_cnt; }
    if (serialized_trie_contains(opts.trie_true, desc.value_begin, value_len) ||
        serialized_trie_contains(opts.trie_false, desc.value_begin, value_len)) {
      atomicAdd(&column_infos[desc.column].bool_count, 1);
    } else if (digit_count == int_req_number_cnt) {
      bool is_negative       = (*desc.value_begin == '-');
      char const *data_begin = desc.value_begin + (is_negative || (*desc.value_begin == '+'));
      cudf::size_type *ptr   = cudf::io::gpu::infer_integral_field_counter(
        data_begin, data_begin + digit_count, is_negative, column_infos[desc.column]);
      atomicAdd(ptr, 1);
    } else if (is_like_float(
                 value_len, digit_count, decimal_count, dash_count + plus_count, exponent_count)) {
      atomicAdd(&column_infos[desc.column].float_count, 1);
    }
    // A date-time field cannot have more than 3 non-special characters
    // A number field cannot have more than one decimal point
    else if (other_count > 3 || decimal_count > 1) {
      atomicAdd(&column_infos[desc.column].string_count, 1);
    } else {
      // A date field can have either one or two '-' or '\'; A legal combination will only have one
      // of them To simplify the process of auto column detection, we are not covering all the
      // date-time formation permutations
      if ((dash_count > 0 && dash_count <= 2 && slash_count == 0) ||
          (dash_count == 0 && slash_count > 0 && slash_count <= 2)) {
        if (colon_count <= 2) {
          atomicAdd(&column_infos[desc.column].datetime_count, 1);
        } else {
          atomicAdd(&column_infos[desc.column].string_count, 1);
        }
      } else {
        // Default field type is string
        atomicAdd(&column_infos[desc.column].string_count, 1);
      }
    }
  }
  if (!are_rows_objects) {
    // For array rows, mark missing fields as null
    for (; input_field_index < num_columns; ++input_field_index)
      atomicAdd(&column_infos[input_field_index].null_count, 1);
  }
}

/**
 * @brief Input data range that contains a field in key:value format.
 */
struct key_value_range {
  char const *key_begin;
  char const *key_end;
  char const *value_begin;
  char const *value_end;
};

/**
 * @brief Parse the next field in key:value format and return ranges of its parts.
 */
__device__ key_value_range get_next_key_value_range(char const *begin,
                                                    char const *end,
                                                    parse_options_view const &opts)
{
  auto const key_range = get_next_key(begin, end, opts.quotechar);

  // Colon between the key and the value
  auto const colon = thrust::find(thrust::seq, key_range.second, end, ':');
  if (colon == end) return {end, end, end};

  // Field value (including delimiters)
  auto const value_end = cudf::io::gpu::seek_field_end(colon + 1, end, opts, true);
  return {key_range.first, key_range.second, colon + 1, value_end};
}

/**
 * @brief Cuda kernel that collects information about JSON object keys in the file.
 *
 * @param[in] options A set of parsing options
 * @param[in] data Input data buffer
 * @param[in] row_offsets The offset of each row in the input
 * @param[out] keys_cnt Number of keys found in the file
 * @param[out] keys_info optional, information (offset, length, hash) for each found key
 */
__global__ void collect_keys_info_kernel(parse_options_view const options,
                                         device_span<char const> const data,
                                         device_span<uint64_t const> const row_offsets,
                                         unsigned long long int *keys_cnt,
                                         thrust::optional<mutable_table_device_view> keys_info)
{
  auto const rec_id = threadIdx.x + (blockDim.x * blockIdx.x);
  if (rec_id >= row_offsets.size()) return;

  auto const row_data_range = get_row_data_range(data, row_offsets, rec_id);

  auto advance = [&](const char *begin) {
    return get_next_key_value_range(begin, row_data_range.second, options);
  };
  for (auto field_range = advance(row_data_range.first);
       field_range.key_begin < row_data_range.second;
       field_range = advance(field_range.value_end)) {
    auto const idx = atomicAdd(keys_cnt, 1);
    if (keys_info.has_value()) {
      auto const len                              = field_range.key_end - field_range.key_begin;
      keys_info->column(0).element<uint64_t>(idx) = field_range.key_begin - data.begin();
      keys_info->column(1).element<uint16_t>(idx) = len;
      keys_info->column(2).element<uint32_t>(idx) =
        MurmurHash3_32<cudf::string_view>{}(cudf::string_view(field_range.key_begin, len));
    }
  }
}

}  // namespace

/**
 * @copydoc cudf::io::json::gpu::convert_json_to_columns
 */
void convert_json_to_columns(parse_options_view const &opts,
                             device_span<char const> const data,
                             device_span<uint64_t const> const row_offsets,
                             device_span<data_type const> const column_types,
                             col_map_type *col_map,
                             device_span<void *const> const output_columns,
                             device_span<bitmask_type *const> const valid_fields,
                             device_span<cudf::size_type> num_valid_fields,
                             rmm::cuda_stream_view stream)
{
  int block_size;
  int min_grid_size;
  CUDA_TRY(hipOccupancyMaxPotentialBlockSize(
    &min_grid_size, &block_size, convert_data_to_columns_kernel));

  const int grid_size = (row_offsets.size() + block_size - 1) / block_size;

  convert_data_to_columns_kernel<<<grid_size, block_size, 0, stream.value()>>>(
    opts, data, row_offsets, column_types, col_map, output_columns, valid_fields, num_valid_fields);

  CUDA_TRY(hipGetLastError());
}

/**
 * @copydoc cudf::io::gpu::detect_data_types
 */

std::vector<cudf::io::column_type_histogram> detect_data_types(
  const parse_options_view &options,
  device_span<char const> const data,
  device_span<uint64_t const> const row_offsets,
  bool do_set_null_count,
  int num_columns,
  col_map_type *col_map,
  rmm::cuda_stream_view stream)
{
  int block_size;
  int min_grid_size;
  CUDA_TRY(
    hipOccupancyMaxPotentialBlockSize(&min_grid_size, &block_size, detect_data_types_kernel));

  rmm::device_vector<cudf::io::column_type_histogram> d_column_infos(
    num_columns, cudf::io::column_type_histogram{});

  if (do_set_null_count) {
    // Set the null count to the row count (all fields assumes to be null).
    thrust::for_each(
      rmm::exec_policy(stream),
      d_column_infos.begin(),
      d_column_infos.end(),
      [num_records = row_offsets.size()] __device__(auto &info) { info.null_count = num_records; });
  }

  // Calculate actual block count to use based on records count
  const int grid_size = (row_offsets.size() + block_size - 1) / block_size;

  detect_data_types_kernel<<<grid_size, block_size, 0, stream.value()>>>(
    options, data, row_offsets, col_map, num_columns, d_column_infos);

  CUDA_TRY(hipGetLastError());

  auto h_column_infos = std::vector<cudf::io::column_type_histogram>(num_columns);

  thrust::copy(d_column_infos.begin(), d_column_infos.end(), h_column_infos.begin());

  return h_column_infos;
}

/**
 * @copydoc cudf::io::json::gpu::gpu_collect_keys_info
 */
void collect_keys_info(parse_options_view const &options,
                       device_span<char const> const data,
                       device_span<uint64_t const> const row_offsets,
                       unsigned long long int *keys_cnt,
                       thrust::optional<mutable_table_device_view> keys_info,
                       rmm::cuda_stream_view stream)
{
  int block_size;
  int min_grid_size;
  CUDA_TRY(
    hipOccupancyMaxPotentialBlockSize(&min_grid_size, &block_size, collect_keys_info_kernel));

  // Calculate actual block count to use based on records count
  const int grid_size = (row_offsets.size() + block_size - 1) / block_size;

  collect_keys_info_kernel<<<grid_size, block_size, 0, stream.value()>>>(
    options, data, row_offsets, keys_cnt, keys_info);

  CUDA_TRY(hipGetLastError());
}

}  // namespace gpu
}  // namespace json
}  // namespace io
}  // namespace cudf
