#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2020, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "json_common.h"
#include "json_gpu.h"

#include <rmm/device_buffer.hpp>

#include <cudf/detail/utilities/trie.cuh>

#include <cudf/utilities/bit.hpp>
#include <cudf/utilities/traits.hpp>
#include <cudf/utilities/type_dispatcher.hpp>

#include <cudf/lists/list_view.cuh>
#include <cudf/strings/string_view.cuh>

#include <io/csv/datetime.cuh>
#include <io/utilities/parsing_utils.cuh>

namespace cudf {
namespace io {
namespace json {
namespace gpu {
using namespace ::cudf;

using string_pair = std::pair<const char *, size_t>;

namespace {
/**
 * @brief CUDA Kernel that modifies the start and stop offsets to exclude
 * the sections outside of the top level brackets.
 *
 * The top level brackets characters are excluded from the resulting range.
 * Parameter stop has the same semantics as end() in STL containers
 * (one past the last element)
 *
 * @param[in] data Pointer to the device buffer containing the data to process
 * @param[in,out] start Offset of the first character in the range
 * @param[in,out] stop Offset of the first character after the range
 *
 * @return void
 **/
__device__ void limit_range_to_brackets(const char *data, long &start, long &stop)
{
  while (start < stop && data[start] != '[' && data[start] != '{') { start++; }
  start++;

  while (start < stop && data[stop - 1] != ']' && data[stop - 1] != '}') { stop--; }
  stop--;
}

/**
 * @brief CUDA kernel that finds the end position of the next field name,
 * including the colon that separates the name from the field value.
 *
 * Returns the position after the colon that precedes the value token.
 *
 * @param[in] data Pointer to the device buffer containing the data to process
 * @param[in] opts Parsing options (e.g. delimiter and quotation character)
 * @param[in] start Offset of the first character in the range
 * @param[in] stop Offset of the first character after the range
 *
 * @return long Position of the first character after the field name.
 **/
__device__ long seek_field_name_end(const char *data,
                                    const ParseOptions opts,
                                    long start,
                                    long stop)
{
  bool quotation = false;
  for (auto pos = start; pos < stop; ++pos) {
    // Ignore escaped quotes
    if (data[pos] == opts.quotechar && data[pos - 1] != '\\') {
      quotation = !quotation;
    } else if (!quotation && data[pos] == ':') {
      return pos + 1;
    }
  }
  return stop;
}

/**
 * @brief Decodes a numeric value base on templated cudf type T with specified
 * base.
 *
 * @param data The character string for parse
 * @param start The index within data to start parsing from
 * @param end The end index within data to end parsing
 * @param opts The global parsing behavior options
 *
 * @return The parsed numeric value
 **/
template <typename T, int base>
__inline__ __device__ T
decode_value(const char *data, long start, long end, ParseOptions const &opts)
{
  return cudf::io::gpu::parse_numeric<T, base>(data, start, end, opts);
}

/**
 * @brief Decodes a numeric value base on templated cudf type T
 *
 * @param data The character string for parse
 * @param start The index within data to start parsing from
 * @param end The end index within data to end parsing
 * @param opts The global parsing behavior options
 *
 * @return The parsed numeric value
 **/
template <typename T>
__inline__ __device__ T
decode_value(const char *data, long start, long end, ParseOptions const &opts)
{
  return cudf::io::gpu::parse_numeric<T>(data, start, end, opts);
}

/**
 * @brief Decodes a timestamp_D
 *
 * @param data The character string for parse
 * @param start The index within data to start parsing from
 * @param end The end index within data to end parsing
 * @param opts The global parsing behavior options
 *
 * @return The parsed timestamp_D
 **/
template <>
__inline__ __device__ cudf::timestamp_D decode_value(const char *data,
                                                     long start,
                                                     long end,
                                                     ParseOptions const &opts)
{
  return parseDateFormat(data, start, end, opts.dayfirst);
}

/**
 * @brief Decodes a timestamp_s
 *
 * @param data The character string for parse
 * @param start The index within data to start parsing from
 * @param end The end index within data to end parsing
 * @param opts The global parsing behavior options
 *
 * @return The parsed timestamp_s
 **/
template <>
__inline__ __device__ cudf::timestamp_s decode_value(const char *data,
                                                     long start,
                                                     long end,
                                                     ParseOptions const &opts)
{
  auto milli = parseDateTimeFormat(data, start, end, opts.dayfirst);
  return milli / 1000;
}

/**
 * @brief Decodes a timestamp_ms
 *
 * @param data The character string for parse
 * @param start The index within data to start parsing from
 * @param end The end index within data to end parsing
 * @param opts The global parsing behavior options
 *
 * @return The parsed timestamp_ms
 **/
template <>
__inline__ __device__ cudf::timestamp_ms decode_value(const char *data,
                                                      long start,
                                                      long end,
                                                      ParseOptions const &opts)
{
  auto milli = parseDateTimeFormat(data, start, end, opts.dayfirst);
  return milli;
}

/**
 * @brief Decodes a timestamp_us
 *
 * @param data The character string for parse
 * @param start The index within data to start parsing from
 * @param end The end index within data to end parsing
 * @param opts The global parsing behavior options
 *
 * @return The parsed timestamp_us
 **/
template <>
__inline__ __device__ cudf::timestamp_us decode_value(const char *data,
                                                      long start,
                                                      long end,
                                                      ParseOptions const &opts)
{
  auto milli = parseDateTimeFormat(data, start, end, opts.dayfirst);
  return milli * 1000;
}

/**
 * @brief Decodes a timestamp_ns
 *
 * @param data The character string for parse
 * @param start The index within data to start parsing from
 * @param end The end index within data to end parsing
 * @param opts The global parsing behavior options
 *
 * @return The parsed timestamp_ns
 **/
template <>
__inline__ __device__ cudf::timestamp_ns decode_value(const char *data,
                                                      long start,
                                                      long end,
                                                      ParseOptions const &opts)
{
  auto milli = parseDateTimeFormat(data, start, end, opts.dayfirst);
  return milli * 1000000;
}

// The purpose of this is merely to allow compilation ONLY
// TODO : make this work for json
#ifndef DURATION_DECODE_VALUE
#define DURATION_DECODE_VALUE(Type)                                   \
  template <>                                                         \
  __inline__ __device__ Type decode_value(                            \
    const char *data, long start, long end, ParseOptions const &opts) \
  {                                                                   \
    return Type{};                                                    \
  }
#endif
DURATION_DECODE_VALUE(duration_D)
DURATION_DECODE_VALUE(duration_s)
DURATION_DECODE_VALUE(duration_ms)
DURATION_DECODE_VALUE(duration_us)
DURATION_DECODE_VALUE(duration_ns)

// The purpose of these is merely to allow compilation ONLY
template <>
__inline__ __device__ cudf::string_view decode_value(const char *data,
                                                     long start,
                                                     long end,
                                                     ParseOptions const &opts)
{
  return cudf::string_view{};
}
template <>
__inline__ __device__ cudf::dictionary32 decode_value(const char *data,
                                                      long start,
                                                      long end,
                                                      ParseOptions const &opts)
{
  return cudf::dictionary32{};
}
template <>
__inline__ __device__ cudf::list_view decode_value(const char *data,
                                                   long start,
                                                   long end,
                                                   ParseOptions const &opts)
{
  return cudf::list_view{};
}

/**
 * @brief Functor for converting plain text data to cuDF data type value.
 **/
struct ConvertFunctor {
  /**
   * @brief Template specialization for operator() for types whose values can be
   * convertible to a 0 or 1 to represent false/true. The converting is done by
   * checking against the default and user-specified true/false values list.
   *
   * It is handled here rather than within convertStrToValue() as that function
   * is used by other types (ex. timestamp) that aren't 'booleable'.
   **/
  template <typename T, typename std::enable_if_t<std::is_integral<T>::value> * = nullptr>
  __host__ __device__ __forceinline__ bool operator()(const char *data,
                                                      void *output_columns,
                                                      long row,
                                                      long start,
                                                      long end,
                                                      const ParseOptions &opts)
  {
    T &value{static_cast<T *>(output_columns)[row]};

    // Check for user-specified true/false values first, where the output is
    // replaced with 1/0 respectively
    const size_t field_len = end - start + 1;
    if (serializedTrieContains(opts.trueValuesTrie, data + start, field_len)) {
      value = 1;
    } else if (serializedTrieContains(opts.falseValuesTrie, data + start, field_len)) {
      value = 0;
    } else {
      value = decode_value<T>(data, start, end, opts);
    }

    return true;
  }

  /**
   * @brief Dispatch for floating points, which are set to NaN if the input
   * is not valid. In such case, the validity mask is set to zero too.
   */
  template <typename T, typename std::enable_if_t<std::is_floating_point<T>::value> * = nullptr>
  __host__ __device__ __forceinline__ bool operator()(
    const char *data, void *out_buffer, size_t row, long start, long end, ParseOptions const &opts)
  {
    auto &value{static_cast<T *>(out_buffer)[row]};
    value = decode_value<T>(data, start, end, opts);
    return !std::isnan(value);
  }

  /**
   * @brief Default template operator() dispatch specialization all data types
   * (including wrapper types) that is not covered by above.
   **/
  template <typename T,
            typename std::enable_if_t<!std::is_floating_point<T>::value and
                                      !std::is_integral<T>::value> * = nullptr>
  __host__ __device__ __forceinline__ bool operator()(const char *data,
                                                      void *output_columns,
                                                      long row,
                                                      long start,
                                                      long end,
                                                      const ParseOptions &opts)
  {
    T &value{static_cast<T *>(output_columns)[row]};
    value = decode_value<T>(data, start, end, opts);

    return true;
  }
};

/**
 * @brief Checks whether the given character is a whitespace character.
 *
 * @param[in] ch The character to check
 *
 * @return True if the input is whitespace, False otherwise
 **/
__inline__ __device__ bool is_whitespace(char ch) { return ch == '\t' || ch == ' '; }

/**
 * @brief Scans a character stream within a range, and adjusts the start and end
 * indices of the range to ignore whitespace and quotation characters.
 *
 * @param[in] data The character stream to scan
 * @param[in,out] start The start index to adjust
 * @param[in,out] end The end index to adjust
 * @param[in] quotechar The character used to denote quotes
 *
 * @return Adjusted or unchanged start_idx and end_idx
 **/
__inline__ __device__ void trim_field_start_end(const char *data,
                                                long *start,
                                                long *end,
                                                char quotechar = '\0')
{
  while ((*start < *end) && is_whitespace(data[*start])) { (*start)++; }
  if ((*start < *end) && data[*start] == quotechar) { (*start)++; }
  while ((*start <= *end) && is_whitespace(data[*end])) { (*end)--; }
  if ((*start <= *end) && data[*end] == quotechar) { (*end)--; }
}

/**
 * @brief Returns true is the input character is a valid digit.
 * Supports both decimal and hexadecimal digits (uppercase and lowercase).
 *
 * @param c Character to check
 * @param is_hex Whether to check as a hexadecimal
 *
 * @return `true` if it is digit-like, `false` otherwise
 */
__device__ __inline__ bool is_digit(char c, bool is_hex = false)
{
  if (c >= '0' && c <= '9') return true;

  if (is_hex) {
    if (c >= 'A' && c <= 'F') return true;
    if (c >= 'a' && c <= 'f') return true;
  }

  return false;
}

/**
 * @brief Returns true if the counters indicate a potentially valid float.
 * False positives are possible because positions are not taken into account.
 * For example, field "e.123-" would match the pattern.
 */
__device__ __inline__ bool is_like_float(
  long len, long digit_cnt, long decimal_cnt, long dash_cnt, long exponent_cnt)
{
  // Can't have more than one exponent and one decimal point
  if (decimal_cnt > 1) return false;
  if (exponent_cnt > 1) return false;
  // Without the exponent or a decimal point, this is an integer, not a float
  if (decimal_cnt == 0 && exponent_cnt == 0) return false;

  // Can only have one '-' per component
  if (dash_cnt > 1 + exponent_cnt) return false;

  // If anything other than these characters is present, it's not a float
  if (digit_cnt + decimal_cnt + dash_cnt + exponent_cnt != len) return false;

  // Needs at least 1 digit, 2 if exponent is present
  if (digit_cnt < 1 + exponent_cnt) return false;

  return true;
}

/**
 * @brief CUDA kernel that parses and converts plain text data into cuDF column data.
 *
 * Data is processed one record at a time
 *
 * @param[in] data The entire data to read
 * @param[in] data_size Size of the data buffer, in bytes
 * @param[in] rec_starts The start of each data record
 * @param[in] num_records The number of lines/rows
 * @param[in] dtypes The data type of each column
 * @param[in] opts A set of parsing options
 * @param[out] output_columns The output column data
 * @param[in] num_columns The number of columns
 * @param[out] valid_fields The bitmaps indicating whether column fields are valid
 * @param[out] num_valid_fields The numbers of valid fields in columns
 *
 * @return void
 **/
__global__ void convert_json_to_columns_kernel(const char *data,
                                               size_t data_size,
                                               const uint64_t *rec_starts,
                                               cudf::size_type num_records,
                                               const data_type *dtypes,
                                               ParseOptions opts,
                                               void *const *output_columns,
                                               int num_columns,
                                               bitmask_type *const *valid_fields,
                                               cudf::size_type *num_valid_fields)
{
  const long rec_id = threadIdx.x + (blockDim.x * blockIdx.x);
  if (rec_id >= num_records) return;

  long start = rec_starts[rec_id];
  // has the same semantics as end() in STL containers (one past last element)
  long stop = ((rec_id < num_records - 1) ? rec_starts[rec_id + 1] : data_size);

  limit_range_to_brackets(data, start, stop);
  const bool is_object = (data[start - 1] == '{');

  for (int col = 0; col < num_columns && start < stop; col++) {
    if (is_object) { start = seek_field_name_end(data, opts, start, stop); }
    // field_end is at the next delimiter/newline
    const long field_end = cudf::io::gpu::seek_field_end(data, opts, start, stop);
    long field_data_last = field_end - 1;
    // Modify start & end to ignore whitespace and quotechars
    trim_field_start_end(data, &start, &field_data_last, opts.quotechar);
    // Empty fields are not legal values
    if (start <= field_data_last &&
        !serializedTrieContains(opts.naValuesTrie, data + start, field_end - start)) {
      // Type dispatcher does not handle strings
      if (dtypes[col].id() == type_id::STRING) {
        auto str_list           = static_cast<string_pair *>(output_columns[col]);
        str_list[rec_id].first  = data + start;
        str_list[rec_id].second = field_data_last - start + 1;

        // set the valid bitmap - all bits were set to 0 to start
        set_bit(valid_fields[col], rec_id);
        atomicAdd(&num_valid_fields[col], 1);
      } else {
        if (cudf::type_dispatcher(dtypes[col],
                                  ConvertFunctor{},
                                  data,
                                  output_columns[col],
                                  rec_id,
                                  start,
                                  field_data_last,
                                  opts)) {
          // set the valid bitmap - all bits were set to 0 to start
          set_bit(valid_fields[col], rec_id);
          atomicAdd(&num_valid_fields[col], 1);
        }
      }
    } else if (dtypes[col].id() == type_id::STRING) {
      auto str_list           = static_cast<string_pair *>(output_columns[col]);
      str_list[rec_id].first  = nullptr;
      str_list[rec_id].second = 0;
    }
    start = field_end + 1;
  }
}

/**
 * @brief CUDA kernel that processes a buffer of data and determines information about the
 * column types within.
 *
 * Data is processed in one row/record at a time, so the number of total
 * threads (tid) is equal to the number of rows.
 *
 * @param[in] data Input data buffer
 * @param[in] data_size Size of the data buffer, in bytes
 * @param[in] opts A set of parsing options
 * @param[in] num_columns The number of columns of input data
 * @param[in] rec_starts The start the input data of interest
 * @param[in] num_records The number of lines/rows of input data
 * @param[out] column_infos The count for each column data type
 *
 * @returns void
 **/
__global__ void detect_json_data_types(const char *data,
                                       size_t data_size,
                                       const ParseOptions opts,
                                       int num_columns,
                                       const uint64_t *rec_starts,
                                       cudf::size_type num_records,
                                       ColumnInfo *column_infos)
{
  long rec_id = threadIdx.x + (blockDim.x * blockIdx.x);
  if (rec_id >= num_records) return;

  long start = rec_starts[rec_id];
  // has the same semantics as end() in STL containers (one past last element)
  long stop = ((rec_id < num_records - 1) ? rec_starts[rec_id + 1] : data_size);

  limit_range_to_brackets(data, start, stop);
  const bool is_object = (data[start - 1] == '{');

  for (int col = 0; col < num_columns; col++) {
    if (is_object) { start = seek_field_name_end(data, opts, start, stop); }
    auto field_start     = start;
    const long field_end = cudf::io::gpu::seek_field_end(data, opts, field_start, stop);
    long field_data_last = field_end - 1;
    trim_field_start_end(data, &field_start, &field_data_last);
    const int field_len = field_data_last - field_start + 1;
    // Advance the start offset
    start = field_end + 1;

    // Checking if the field is empty
    if (field_start > field_data_last ||
        serializedTrieContains(opts.naValuesTrie, data + field_start, field_len)) {
      atomicAdd(&column_infos[col].null_count, 1);
      continue;
    }
    // Don't need counts to detect strings, any field in quotes is deduced to be a string
    if (data[field_start] == opts.quotechar && data[field_data_last] == opts.quotechar) {
      atomicAdd(&column_infos[col].string_count, 1);
      continue;
    }

    int digit_count    = 0;
    int decimal_count  = 0;
    int slash_count    = 0;
    int dash_count     = 0;
    int colon_count    = 0;
    int exponent_count = 0;
    int other_count    = 0;

    const bool maybe_hex =
      ((field_len > 2 && data[field_start] == '0' && data[field_start + 1] == 'x') ||
       (field_len > 3 && data[field_start] == '-' && data[field_start + 1] == '0' &&
        data[field_start + 2] == 'x'));
    for (long pos = field_start; pos <= field_data_last; pos++) {
      if (is_digit(data[pos], maybe_hex)) {
        digit_count++;
        continue;
      }
      // Looking for unique characters that will help identify column types
      switch (data[pos]) {
        case '.': decimal_count++; break;
        case '-': dash_count++; break;
        case '/': slash_count++; break;
        case ':': colon_count++; break;
        case 'e':
        case 'E':
          if (!maybe_hex && pos > field_start && pos < field_data_last) exponent_count++;
          break;
        default: other_count++; break;
      }
    }

    // Integers have to have the length of the string
    int int_req_number_cnt = field_len;
    // Off by one if they start with a minus sign
    if (data[field_start] == '-' && field_len > 1) { --int_req_number_cnt; }
    // Off by one if they are a hexadecimal number
    if (maybe_hex) { --int_req_number_cnt; }
    if (serializedTrieContains(opts.trueValuesTrie, data + field_start, field_len) ||
        serializedTrieContains(opts.falseValuesTrie, data + field_start, field_len)) {
      atomicAdd(&column_infos[col].bool_count, 1);
    } else if (digit_count == int_req_number_cnt) {
      atomicAdd(&column_infos[col].int_count, 1);
    } else if (is_like_float(field_len, digit_count, decimal_count, dash_count, exponent_count)) {
      atomicAdd(&column_infos[col].float_count, 1);
    }
    // A date-time field cannot have more than 3 non-special characters
    // A number field cannot have more than one decimal point
    else if (other_count > 3 || decimal_count > 1) {
      atomicAdd(&column_infos[col].string_count, 1);
    } else {
      // A date field can have either one or two '-' or '\'; A legal combination will only have one
      // of them To simplify the process of auto column detection, we are not covering all the
      // date-time formation permutations
      if ((dash_count > 0 && dash_count <= 2 && slash_count == 0) ||
          (dash_count == 0 && slash_count > 0 && slash_count <= 2)) {
        if (colon_count <= 2) {
          atomicAdd(&column_infos[col].datetime_count, 1);
        } else {
          atomicAdd(&column_infos[col].string_count, 1);
        }
      } else {
        // Default field type is string
        atomicAdd(&column_infos[col].string_count, 1);
      }
    }
  }
}

}  // namespace

/**
 * @copydoc cudf::io::json::gpu::convert_json_to_columns
 *
 **/
void convert_json_to_columns(rmm::device_buffer const &input_data,
                             data_type *const dtypes,
                             void *const *output_columns,
                             cudf::size_type num_records,
                             cudf::size_type num_columns,
                             const uint64_t *rec_starts,
                             bitmask_type *const *valid_fields,
                             cudf::size_type *num_valid_fields,
                             ParseOptions const &opts,
                             hipStream_t stream)
{
  int block_size;
  int min_grid_size;
  CUDA_TRY(hipOccupancyMaxPotentialBlockSize(
    &min_grid_size, &block_size, convert_json_to_columns_kernel));

  const int grid_size = (num_records + block_size - 1) / block_size;

  convert_json_to_columns_kernel<<<grid_size, block_size, 0, stream>>>(
    static_cast<const char *>(input_data.data()),
    input_data.size(),
    rec_starts,
    num_records,
    dtypes,
    opts,
    output_columns,
    num_columns,
    valid_fields,
    num_valid_fields);

  CUDA_TRY(hipGetLastError());
}

/**
 * @copydoc cudf::io::json::gpu::detect_data_types
 *
 **/
void detect_data_types(ColumnInfo *column_infos,
                       const char *data,
                       size_t data_size,
                       const ParseOptions &options,
                       int num_columns,
                       const uint64_t *rec_starts,
                       cudf::size_type num_records,
                       hipStream_t stream)
{
  int block_size;
  int min_grid_size;
  CUDA_TRY(hipOccupancyMaxPotentialBlockSize(&min_grid_size, &block_size, detect_json_data_types));

  // Calculate actual block count to use based on records count
  const int grid_size = (num_records + block_size - 1) / block_size;

  detect_json_data_types<<<grid_size, block_size, 0, stream>>>(
    data, data_size, options, num_columns, rec_starts, num_records, column_infos);

  CUDA_TRY(hipGetLastError());
}

}  // namespace gpu
}  // namespace json
}  // namespace io
}  // namespace cudf
