#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2022, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "nested_json.hpp"
#include <hash/hash_allocator.cuh>
#include <hash/helper_functions.cuh>
#include <io/utilities/hostdevice_vector.hpp>

#include <cudf/detail/hashing.hpp>
#include <cudf/detail/nvtx/ranges.hpp>
#include <cudf/detail/scatter.cuh>
#include <cudf/detail/utilities/hash_functions.cuh>
#include <cudf/detail/utilities/vector_factories.hpp>
#include <cudf/utilities/error.hpp>
#include <cudf/utilities/span.hpp>

#include <cuco/static_map.cuh>

#include <cub/device/device_radix_sort.cuh>

#include <rmm/cuda_stream_view.hpp>
#include <rmm/device_uvector.hpp>
#include <rmm/exec_policy.hpp>
#include <rmm/mr/device/polymorphic_allocator.hpp>

#include <thrust/binary_search.h>
#include <thrust/copy.h>
#include <thrust/count.h>
#include <thrust/fill.h>
#include <thrust/gather.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/iterator/discard_iterator.h>
#include <thrust/iterator/permutation_iterator.h>
#include <thrust/iterator/transform_output_iterator.h>
#include <thrust/iterator/zip_iterator.h>
#include <thrust/reduce.h>
#include <thrust/remove.h>
#include <thrust/scan.h>
#include <thrust/sequence.h>
#include <thrust/sort.h>
#include <thrust/tabulate.h>
#include <thrust/transform.h>

#include <limits>

namespace cudf::io::json {
namespace detail {
namespace {

/**
 * @brief Utility for calling thrust::copy_if
 *
 * Workaround for thrust::copy_if bug (https://github.com/NVIDIA/thrust/issues/1302)
 * where it cannot iterate over int-max values `distance(first,last) > int-max`
 * This calls thrust::copy_if in 2B chunks instead.
 */
template <typename InputIterator,
          typename StencilIterator,
          typename OutputIterator,
          typename Predicate>
OutputIterator thrust_copy_if(rmm::exec_policy policy,
                              InputIterator first,
                              InputIterator last,
                              StencilIterator stencil,
                              OutputIterator result,
                              Predicate pred)
{
  auto const copy_size = std::min(static_cast<std::size_t>(std::distance(first, last)),
                                  static_cast<std::size_t>(std::numeric_limits<int>::max()));

  auto itr = first;
  while (itr != last) {
    auto const copy_end =
      static_cast<std::size_t>(std::distance(itr, last)) <= copy_size ? last : itr + copy_size;
    result = thrust::copy_if(policy, itr, copy_end, stencil, result, pred);
    stencil += std::distance(itr, copy_end);
    itr = copy_end;
  }
  return result;
}

template <typename InputIterator, typename OutputIterator, typename Predicate>
OutputIterator thrust_copy_if(rmm::exec_policy policy,
                              InputIterator first,
                              InputIterator last,
                              OutputIterator result,
                              Predicate pred)
{
  return thrust_copy_if(policy, first, last, first, result, pred);
}
}  // namespace

// The node that a token represents
struct token_to_node {
  __device__ auto operator()(PdaTokenT const token) -> NodeT
  {
    switch (token) {
      case token_t::StructBegin: return NC_STRUCT;
      case token_t::ListBegin: return NC_LIST;
      case token_t::StringBegin: return NC_STR;
      case token_t::ValueBegin:
        return NC_STR;  // NC_VAL;
      // NV_VAL is removed because type inference and
      // reduce_to_column_tree category collapsing takes care of this.
      case token_t::FieldNameBegin: return NC_FN;
      default: return NC_ERR;
    };
  }
};

// Convert token indices to node range for each valid node.
struct node_ranges {
  device_span<PdaTokenT const> tokens;
  device_span<SymbolOffsetT const> token_indices;
  bool include_quote_char;
  __device__ auto operator()(size_type i) -> thrust::tuple<SymbolOffsetT, SymbolOffsetT>
  {
    // Whether a token expects to be followed by its respective end-of-* token partner
    auto const is_begin_of_section = [] __device__(PdaTokenT const token) {
      switch (token) {
        case token_t::StringBegin:
        case token_t::ValueBegin:
        case token_t::FieldNameBegin: return true;
        default: return false;
      };
    };
    // The end-of-* partner token for a given beginning-of-* token
    auto const end_of_partner = [] __device__(PdaTokenT const token) {
      switch (token) {
        case token_t::StringBegin: return token_t::StringEnd;
        case token_t::ValueBegin: return token_t::ValueEnd;
        case token_t::FieldNameBegin: return token_t::FieldNameEnd;
        default: return token_t::ErrorBegin;
      };
    };
    // Includes quote char for end-of-string token or Skips the quote char for
    // beginning-of-field-name token
    auto const get_token_index = [include_quote_char = include_quote_char] __device__(
                                   PdaTokenT const token, SymbolOffsetT const token_index) {
      constexpr SymbolOffsetT quote_char_size = 1;
      switch (token) {
        // Strip off quote char included for StringBegin
        case token_t::StringBegin: return token_index + (include_quote_char ? 0 : quote_char_size);
        // Strip off or Include trailing quote char for string values for StringEnd
        case token_t::StringEnd: return token_index + (include_quote_char ? quote_char_size : 0);
        // Strip off quote char included for FieldNameBegin
        case token_t::FieldNameBegin: return token_index + quote_char_size;
        default: return token_index;
      };
    };
    PdaTokenT const token = tokens[i];
    // The section from the original JSON input that this token demarcates
    SymbolOffsetT range_begin = get_token_index(token, token_indices[i]);
    SymbolOffsetT range_end   = range_begin + 1;  // non-leaf, non-field nodes ignore this value.
    if (is_begin_of_section(token)) {
      if ((i + 1) < tokens.size() && end_of_partner(token) == tokens[i + 1]) {
        // Update the range_end for this pair of tokens
        range_end = get_token_index(tokens[i + 1], token_indices[i + 1]);
      }
    }
    return thrust::make_tuple(range_begin, range_end);
  }
};

/**
 * @brief Returns stable sorted keys and its sorted order
 *
 * Uses cub stable radix sort. The order is internally generated, hence it saves a copy and memory.
 * Since the key and order is returned, using double buffer helps to avoid extra copy to user
 * provided output iterator.
 *
 * @tparam IndexType sorted order type
 * @tparam KeyType key type
 * @param keys keys to sort
 * @param stream CUDA stream used for device memory operations and kernel launches.
 * @return Sorted keys and indices producing that sorted order
 */
template <typename IndexType = size_t, typename KeyType>
std::pair<rmm::device_uvector<KeyType>, rmm::device_uvector<IndexType>> stable_sorted_key_order(
  cudf::device_span<KeyType const> keys, rmm::cuda_stream_view stream)
{
  CUDF_FUNC_RANGE();

  // Determine temporary device storage requirements
  rmm::device_uvector<KeyType> keys_buffer1(keys.size(), stream);
  rmm::device_uvector<KeyType> keys_buffer2(keys.size(), stream);
  rmm::device_uvector<IndexType> order_buffer1(keys.size(), stream);
  rmm::device_uvector<IndexType> order_buffer2(keys.size(), stream);
  hipcub::DoubleBuffer<IndexType> order_buffer(order_buffer1.data(), order_buffer2.data());
  hipcub::DoubleBuffer<KeyType> keys_buffer(keys_buffer1.data(), keys_buffer2.data());
  size_t temp_storage_bytes = 0;
  hipcub::DeviceRadixSort::SortPairs(
    nullptr, temp_storage_bytes, keys_buffer, order_buffer, keys.size());
  rmm::device_buffer d_temp_storage(temp_storage_bytes, stream);

  thrust::copy(rmm::exec_policy(stream), keys.begin(), keys.end(), keys_buffer1.begin());
  thrust::sequence(rmm::exec_policy(stream), order_buffer1.begin(), order_buffer1.end());

  hipcub::DeviceRadixSort::SortPairs(d_temp_storage.data(),
                                  temp_storage_bytes,
                                  keys_buffer,
                                  order_buffer,
                                  keys.size(),
                                  0,
                                  sizeof(KeyType) * 8,
                                  stream.value());

  return std::pair{keys_buffer.Current() == keys_buffer1.data() ? std::move(keys_buffer1)
                                                                : std::move(keys_buffer2),
                   order_buffer.Current() == order_buffer1.data() ? std::move(order_buffer1)
                                                                  : std::move(order_buffer2)};
}

/**
 * @brief Propagate parent node to siblings from first sibling.
 *
 * @param node_levels Node levels of each node
 * @param parent_node_ids parent node ids initialized for first child of each push node,
 *                       and other siblings are initialized to -1.
 * @param stream CUDA stream used for device memory operations and kernel launches.
 */
void propagate_parent_to_siblings(cudf::device_span<TreeDepthT const> node_levels,
                                  cudf::device_span<NodeIndexT> parent_node_ids,
                                  rmm::cuda_stream_view stream)
{
  CUDF_FUNC_RANGE();
  auto [sorted_node_levels, sorted_order] = stable_sorted_key_order<size_type>(node_levels, stream);
  // instead of gather, using permutation_iterator, which is ~17% faster

  thrust::inclusive_scan_by_key(
    rmm::exec_policy(stream),
    sorted_node_levels.begin(),
    sorted_node_levels.end(),
    thrust::make_permutation_iterator(parent_node_ids.begin(), sorted_order.begin()),
    thrust::make_permutation_iterator(parent_node_ids.begin(), sorted_order.begin()),
    thrust::equal_to<TreeDepthT>{},
    thrust::maximum<NodeIndexT>{});
}

// Generates a tree representation of the given tokens, token_indices.
tree_meta_t get_tree_representation(device_span<PdaTokenT const> tokens,
                                    device_span<SymbolOffsetT const> token_indices,
                                    rmm::cuda_stream_view stream,
                                    rmm::mr::device_memory_resource* mr)
{
  CUDF_FUNC_RANGE();
  // Whether a token does represent a node in the tree representation
  auto const is_node = [] __device__(PdaTokenT const token) -> bool {
    switch (token) {
      case token_t::StructBegin:
      case token_t::ListBegin:
      case token_t::StringBegin:
      case token_t::ValueBegin:
      case token_t::FieldNameBegin:
      case token_t::ErrorBegin: return true;
      default: return false;
    };
  };

  // Whether the token pops from the parent node stack
  auto const does_pop = [] __device__(PdaTokenT const token) -> bool {
    switch (token) {
      case token_t::StructMemberEnd:
      case token_t::StructEnd:
      case token_t::ListEnd: return true;
      default: return false;
    };
  };

  // Whether the token pushes onto the parent node stack
  auto const does_push = [] __device__(PdaTokenT const token) -> bool {
    switch (token) {
      case token_t::FieldNameBegin:
      case token_t::StructBegin:
      case token_t::ListBegin: return true;
      default: return false;
    };
  };

  // Look for ErrorBegin and report the point of error.
  if (auto const error_count =
        thrust::count(rmm::exec_policy(stream), tokens.begin(), tokens.end(), token_t::ErrorBegin);
      error_count > 0) {
    auto const error_location =
      thrust::find(rmm::exec_policy(stream), tokens.begin(), tokens.end(), token_t::ErrorBegin);
    SymbolOffsetT error_index;
    CUDF_CUDA_TRY(
      hipMemcpyAsync(&error_index,
                      token_indices.data() + thrust::distance(tokens.begin(), error_location),
                      sizeof(SymbolOffsetT),
                      hipMemcpyDeviceToHost,
                      stream.value()));
    stream.synchronize();
    CUDF_FAIL("JSON Parser encountered an invalid format at location " +
              std::to_string(error_index));
  }

  auto const num_tokens = tokens.size();
  auto const num_nodes =
    thrust::count_if(rmm::exec_policy(stream), tokens.begin(), tokens.end(), is_node);

  // Node levels: transform_exclusive_scan, copy_if.
  rmm::device_uvector<TreeDepthT> node_levels(num_nodes, stream, mr);
  {
    rmm::device_uvector<TreeDepthT> token_levels(num_tokens, stream);
    auto const push_pop_it = thrust::make_transform_iterator(
      tokens.begin(), [does_push, does_pop] __device__(PdaTokenT const token) -> size_type {
        return does_push(token) - does_pop(token);
      });
    thrust::exclusive_scan(
      rmm::exec_policy(stream), push_pop_it, push_pop_it + num_tokens, token_levels.begin());

    auto const node_levels_end = thrust_copy_if(rmm::exec_policy(stream),
                                                token_levels.begin(),
                                                token_levels.end(),
                                                tokens.begin(),
                                                node_levels.begin(),
                                                is_node);
    CUDF_EXPECTS(thrust::distance(node_levels.begin(), node_levels_end) == num_nodes,
                 "node level count mismatch");
  }

  // Node parent ids:
  // previous push node_id transform, stable sort by level, segmented scan with Max, reorder.
  rmm::device_uvector<NodeIndexT> parent_node_ids(num_nodes, stream, mr);
  // This block of code is generalized logical stack algorithm. TODO: make this a separate function.
  {
    rmm::device_uvector<NodeIndexT> node_token_ids(num_nodes, stream);
    thrust_copy_if(rmm::exec_policy(stream),
                   thrust::make_counting_iterator<NodeIndexT>(0),
                   thrust::make_counting_iterator<NodeIndexT>(0) + num_tokens,
                   tokens.begin(),
                   node_token_ids.begin(),
                   is_node);

    // previous push node_id
    // if previous node is a push, then i-1
    // if previous node is FE, then i-2 (returns FB's index)
    // if previous node is SMB and its previous node is a push, then i-2
    // eg. `{ SMB FB FE VB VE SME` -> `{` index as FB's parent.
    // else -1
    auto const first_childs_parent_token_id = [tokens_gpu =
                                                 tokens.begin()] __device__(auto i) -> NodeIndexT {
      if (i <= 0) { return -1; }
      if (tokens_gpu[i - 1] == token_t::StructBegin or tokens_gpu[i - 1] == token_t::ListBegin) {
        return i - 1;
      } else if (tokens_gpu[i - 1] == token_t::FieldNameEnd) {
        return i - 2;
      } else if (tokens_gpu[i - 1] == token_t::StructMemberBegin and
                 (tokens_gpu[i - 2] == token_t::StructBegin ||
                  tokens_gpu[i - 2] == token_t::ListBegin)) {
        return i - 2;
      } else {
        return -1;
      }
    };

    thrust::transform(
      rmm::exec_policy(stream),
      node_token_ids.begin(),
      node_token_ids.end(),
      parent_node_ids.begin(),
      [node_ids_gpu = node_token_ids.begin(), num_nodes, first_childs_parent_token_id] __device__(
        NodeIndexT const tid) -> NodeIndexT {
        auto const pid = first_childs_parent_token_id(tid);
        return pid < 0
                 ? parent_node_sentinel
                 : thrust::lower_bound(thrust::seq, node_ids_gpu, node_ids_gpu + num_nodes, pid) -
                     node_ids_gpu;
        // parent_node_sentinel is -1, useful for segmented max operation below
      });
  }
  // Propagate parent node to siblings from first sibling - inplace.
  propagate_parent_to_siblings(
    cudf::device_span<TreeDepthT const>{node_levels.data(), node_levels.size()},
    parent_node_ids,
    stream);

  // Node categories: copy_if with transform.
  rmm::device_uvector<NodeT> node_categories(num_nodes, stream, mr);
  auto const node_categories_it =
    thrust::make_transform_output_iterator(node_categories.begin(), token_to_node{});
  auto const node_categories_end = thrust_copy_if(
    rmm::exec_policy(stream), tokens.begin(), tokens.end(), node_categories_it, is_node);
  CUDF_EXPECTS(node_categories_end - node_categories_it == num_nodes,
               "node category count mismatch");

  // Node ranges: copy_if with transform.
  rmm::device_uvector<SymbolOffsetT> node_range_begin(num_nodes, stream, mr);
  rmm::device_uvector<SymbolOffsetT> node_range_end(num_nodes, stream, mr);
  auto const node_range_tuple_it =
    thrust::make_zip_iterator(node_range_begin.begin(), node_range_end.begin());
  // Whether the tokenizer stage should keep quote characters for string values
  // If the tokenizer keeps the quote characters, they may be stripped during type casting
  constexpr bool include_quote_char = true;
  auto const node_range_out_it      = thrust::make_transform_output_iterator(
    node_range_tuple_it, node_ranges{tokens, token_indices, include_quote_char});

  auto const node_range_out_end =
    thrust_copy_if(rmm::exec_policy(stream),
                   thrust::make_counting_iterator<size_type>(0),
                   thrust::make_counting_iterator<size_type>(0) + num_tokens,
                   node_range_out_it,
                   [is_node, tokens_gpu = tokens.begin()] __device__(size_type i) -> bool {
                     return is_node(tokens_gpu[i]);
                   });
  CUDF_EXPECTS(node_range_out_end - node_range_out_it == num_nodes, "node range count mismatch");

  return {std::move(node_categories),
          std::move(parent_node_ids),
          std::move(node_levels),
          std::move(node_range_begin),
          std::move(node_range_end)};
}

/**
 * @brief Generates unique node_type id for each node.
 * Field nodes with the same name are assigned the same node_type id.
 * List, Struct, and String nodes are assigned their category values as node_type ids.
 *
 * All inputs and outputs are in node_id order.
 * @param d_input JSON string in device memory
 * @param d_tree Tree representation of the JSON
 * @param stream CUDA stream used for device memory operations and kernel launches.
 * @return Vector of node_type ids
 */
rmm::device_uvector<size_type> hash_node_type_with_field_name(device_span<SymbolT const> d_input,
                                                              tree_meta_t const& d_tree,
                                                              rmm::cuda_stream_view stream)
{
  CUDF_FUNC_RANGE();
  using hash_table_allocator_type = rmm::mr::stream_allocator_adaptor<default_allocator<char>>;
  using hash_map_type =
    cuco::static_map<size_type, size_type, cuda::thread_scope_device, hash_table_allocator_type>;

  auto const num_nodes  = d_tree.node_categories.size();
  auto const num_fields = thrust::count(rmm::exec_policy(stream),
                                        d_tree.node_categories.begin(),
                                        d_tree.node_categories.end(),
                                        node_t::NC_FN);

  constexpr size_type empty_node_index_sentinel = -1;
  hash_map_type key_map{compute_hash_table_size(num_fields, 40),  // 40% occupancy in hash map
                        cuco::sentinel::empty_key{empty_node_index_sentinel},
                        cuco::sentinel::empty_value{empty_node_index_sentinel},
                        hash_table_allocator_type{default_allocator<char>{}, stream},
                        stream.value()};
  auto const d_hasher = [d_input          = d_input.data(),
                         node_range_begin = d_tree.node_range_begin.data(),
                         node_range_end   = d_tree.node_range_end.data()] __device__(auto node_id) {
    auto const field_name = cudf::string_view(d_input + node_range_begin[node_id],
                                              node_range_end[node_id] - node_range_begin[node_id]);
    return cudf::detail::default_hash<cudf::string_view>{}(field_name);
  };
  auto const d_equal = [d_input          = d_input.data(),
                        node_range_begin = d_tree.node_range_begin.data(),
                        node_range_end   = d_tree.node_range_end.data()] __device__(auto node_id1,
                                                                                  auto node_id2) {
    auto const field_name1 = cudf::string_view(
      d_input + node_range_begin[node_id1], node_range_end[node_id1] - node_range_begin[node_id1]);
    auto const field_name2 = cudf::string_view(
      d_input + node_range_begin[node_id2], node_range_end[node_id2] - node_range_begin[node_id2]);
    return field_name1 == field_name2;
  };
  // key-value pairs: uses node_id itself as node_type. (unique node_id for a field name due to
  // hashing)
  auto const iter = cudf::detail::make_counting_transform_iterator(
    0, [] __device__(size_type i) { return cuco::make_pair(i, i); });

  auto const is_field_name_node = [node_categories =
                                     d_tree.node_categories.data()] __device__(auto node_id) {
    return node_categories[node_id] == node_t::NC_FN;
  };
  key_map.insert_if(iter,
                    iter + num_nodes,
                    thrust::counting_iterator<size_type>(0),  // stencil
                    is_field_name_node,
                    d_hasher,
                    d_equal,
                    stream.value());

  auto const get_hash_value =
    [key_map = key_map.get_device_view(), d_hasher, d_equal] __device__(auto node_id) -> size_type {
    auto const it = key_map.find(node_id, d_hasher, d_equal);
    return (it == key_map.end()) ? size_type{0} : it->second.load(cuda::std::memory_order_relaxed);
  };

  // convert field nodes to node indices, and other nodes to enum value.
  rmm::device_uvector<size_type> node_type(num_nodes, stream);
  thrust::tabulate(rmm::exec_policy(stream),
                   node_type.begin(),
                   node_type.end(),
                   [node_categories = d_tree.node_categories.data(),
                    is_field_name_node,
                    get_hash_value] __device__(auto node_id) -> size_type {
                     if (is_field_name_node(node_id))
                       return static_cast<size_type>(NUM_NODE_CLASSES) + get_hash_value(node_id);
                     else
                       return static_cast<size_type>(node_categories[node_id]);
                   });
  return node_type;
}

// Two level hashing algorithm
// 1. Convert node_category+fieldname to node_type. (passed as argument)
//   a. Create a hashmap to hash field name and assign unique node id as values.
//   b. Convert the node categories to node types.
//      Node type is defined as node category enum value if it is not a field node,
//      otherwise it is the unique node id assigned by the hashmap (value shifted by #NUM_CATEGORY).
// 2. Set operation on entire path of each node
//   a. Create a hash map with hash of {node_level, node_type} of its node and the entire parent
//      until root.
//   b. While creating hashmap, transform node id to unique node ids that are inserted into the
//      hash map. This mimics set operation with hash map. This unique node ids are set ids.
//   c. Return this converted set ids, which are the hash map keys/values, and unique set ids.
std::pair<rmm::device_uvector<size_type>, rmm::device_uvector<size_type>> hash_node_path(
  device_span<TreeDepthT const> node_levels,
  device_span<size_type const> node_type,
  device_span<NodeIndexT const> parent_node_ids,
  rmm::cuda_stream_view stream,
  rmm::mr::device_memory_resource* mr)
{
  CUDF_FUNC_RANGE();
  auto const num_nodes = parent_node_ids.size();
  rmm::device_uvector<size_type> col_id(num_nodes, stream, mr);

  using hash_table_allocator_type = rmm::mr::stream_allocator_adaptor<default_allocator<char>>;
  using hash_map_type =
    cuco::static_map<size_type, size_type, cuda::thread_scope_device, hash_table_allocator_type>;

  constexpr size_type empty_node_index_sentinel = -1;
  hash_map_type key_map{compute_hash_table_size(num_nodes),  // TODO reduce oversubscription
                        cuco::sentinel::empty_key{empty_node_index_sentinel},
                        cuco::sentinel::empty_value{empty_node_index_sentinel},
                        cuco::sentinel::erased_key{-2},
                        hash_table_allocator_type{default_allocator<char>{}, stream},
                        stream.value()};
  // path compression is not used since extra writes make all map operations slow.
  auto const d_hasher = [node_level      = node_levels.begin(),
                         node_type       = node_type.begin(),
                         parent_node_ids = parent_node_ids.begin()] __device__(auto node_id) {
    auto hash =
      cudf::detail::hash_combine(cudf::detail::default_hash<TreeDepthT>{}(node_level[node_id]),
                                 cudf::detail::default_hash<size_type>{}(node_type[node_id]));
    node_id = parent_node_ids[node_id];
    while (node_id != parent_node_sentinel) {
      hash = cudf::detail::hash_combine(
        hash, cudf::detail::default_hash<TreeDepthT>{}(node_level[node_id]));
      hash = cudf::detail::hash_combine(
        hash, cudf::detail::default_hash<size_type>{}(node_type[node_id]));
      node_id = parent_node_ids[node_id];
    }
    return hash;
  };

  rmm::device_uvector<hash_value_type> node_hash(num_nodes, stream);
  thrust::tabulate(rmm::exec_policy(stream), node_hash.begin(), node_hash.end(), d_hasher);
  auto const d_hashed_cache = [node_hash = node_hash.begin()] __device__(auto node_id) {
    return node_hash[node_id];
  };

  auto const d_equal = [node_level      = node_levels.begin(),
                        node_type       = node_type.begin(),
                        parent_node_ids = parent_node_ids.begin(),
                        d_hashed_cache] __device__(auto node_id1, auto node_id2) {
    if (node_id1 == node_id2) return true;
    if (d_hashed_cache(node_id1) != d_hashed_cache(node_id2)) return false;
    auto const is_equal_level = [node_level, node_type](auto node_id1, auto node_id2) {
      if (node_id1 == node_id2) return true;
      return node_level[node_id1] == node_level[node_id2] and
             node_type[node_id1] == node_type[node_id2];
    };
    // if both nodes have same node types at all levels, it will check until it has common parent
    // or root.
    while (node_id1 != parent_node_sentinel and node_id2 != parent_node_sentinel and
           node_id1 != node_id2 and is_equal_level(node_id1, node_id2)) {
      node_id1 = parent_node_ids[node_id1];
      node_id2 = parent_node_ids[node_id2];
    }
    return node_id1 == node_id2;
  };

  // insert and convert node ids to unique set ids
  auto const num_inserted = thrust::count_if(
    rmm::exec_policy(stream),
    thrust::make_counting_iterator<size_type>(0),
    thrust::make_counting_iterator<size_type>(num_nodes),
    [d_hashed_cache,
     d_equal,
     view       = key_map.get_device_mutable_view(),
     uq_node_id = col_id.begin()] __device__(auto node_id) mutable {
      auto it = view.insert_and_find(cuco::make_pair(node_id, node_id), d_hashed_cache, d_equal);
      uq_node_id[node_id] = (it.first)->first.load(cuda::std::memory_order_relaxed);
      return it.second;
    });

  auto const num_columns = num_inserted;  // key_map.get_size() is not updated.
  rmm::device_uvector<size_type> unique_keys(num_columns, stream);
  key_map.retrieve_all(unique_keys.begin(), thrust::make_discard_iterator(), stream.value());

  return {std::move(col_id), std::move(unique_keys)};
}

/**
 * @brief Generates column id and parent column id for each node
 *
 * 1. Generate col_id:
 *    a. Set operation on entire path of each node, translate each node id to set id.
 *       (two level hashing)
 *    b. gather unique set ids.
 *    c. sort and use binary search to generate column ids.
 *    d. Translate parent node ids to parent column ids.
 *
 * All inputs and outputs are in node_id order.
 * @param d_input JSON string in device memory
 * @param d_tree Tree representation of the JSON
 * @param stream CUDA stream used for device memory operations and kernel launches
 * @param mr Device memory resource used to allocate the returned column's device memory
 * @return column_id, parent_column_id
 */
std::pair<rmm::device_uvector<NodeIndexT>, rmm::device_uvector<NodeIndexT>> generate_column_id(
  device_span<SymbolT const> d_input,
  tree_meta_t const& d_tree,
  rmm::cuda_stream_view stream,
  rmm::mr::device_memory_resource* mr)
{
  CUDF_FUNC_RANGE();
  auto const num_nodes = d_tree.node_categories.size();

  // Two level hashing:
  //   one for field names -> node_type and,
  //   another for {node_level, node_category} + field hash for the entire path
  //    which is {node_level, node_type} recursively using parent_node_id
  auto [col_id, unique_keys] = [&]() {
    // Convert node_category + field_name to node_type.
    rmm::device_uvector<size_type> node_type =
      hash_node_type_with_field_name(d_input, d_tree, stream);

    // hash entire path from node to root.
    return hash_node_path(d_tree.node_levels, node_type, d_tree.parent_node_ids, stream, mr);
  }();

  thrust::sort(rmm::exec_policy(stream), unique_keys.begin(), unique_keys.end());
  thrust::lower_bound(rmm::exec_policy(stream),
                      unique_keys.begin(),
                      unique_keys.end(),
                      col_id.begin(),
                      col_id.end(),
                      col_id.begin());

  rmm::device_uvector<size_type> parent_col_id(num_nodes, stream, mr);
  thrust::transform(rmm::exec_policy(stream),
                    d_tree.parent_node_ids.begin(),
                    d_tree.parent_node_ids.end(),
                    parent_col_id.begin(),
                    [col_id = col_id.begin()] __device__(auto node_id) {
                      return node_id >= 0 ? col_id[node_id] : parent_node_sentinel;
                    });

  return {std::move(col_id), std::move(parent_col_id)};
}

/**
 * @brief Computes row indices of each node in the hierarchy.
 * 2. Generate row_offset.
 *   a. Extract only list children
 *   b. stable_sort by parent_col_id.
 *   c. scan_by_key {parent_col_id} (done only on nodes who's parent is list)
 *   d. propagate to non-list leaves from parent list node by recursion
 *
 * pre-condition:
 *  d_tree.node_categories, d_tree.parent_node_ids, parent_col_id are in order of node_id.
 * post-condition: row_offsets is in order of node_id.
 *  parent_col_id is moved and reused inside this function.
 * @param parent_col_id parent node's column id
 * @param d_tree Tree representation of the JSON string
 * @param stream CUDA stream used for device memory operations and kernel launches.
 * @param mr Device memory resource used to allocate the returned column's device memory.
 * @return row_offsets
 */
rmm::device_uvector<size_type> compute_row_offsets(rmm::device_uvector<NodeIndexT>&& parent_col_id,
                                                   tree_meta_t const& d_tree,
                                                   rmm::cuda_stream_view stream,
                                                   rmm::mr::device_memory_resource* mr)
{
  CUDF_FUNC_RANGE();
  auto const num_nodes = d_tree.node_categories.size();

  rmm::device_uvector<size_type> scatter_indices(num_nodes, stream);
  thrust::sequence(rmm::exec_policy(stream), scatter_indices.begin(), scatter_indices.end());

  // Extract only list children. (nodes who's parent is a list/root)
  auto const list_parent_end =
    thrust::remove_if(rmm::exec_policy(stream),
                      thrust::make_zip_iterator(parent_col_id.begin(), scatter_indices.begin()),
                      thrust::make_zip_iterator(parent_col_id.end(), scatter_indices.end()),
                      d_tree.parent_node_ids.begin(),
                      [node_categories = d_tree.node_categories.begin()] __device__(auto pnid) {
                        return !(pnid == parent_node_sentinel || node_categories[pnid] == NC_LIST);
                      });
  auto const num_list_parent = thrust::distance(
    thrust::make_zip_iterator(parent_col_id.begin(), scatter_indices.begin()), list_parent_end);

  thrust::stable_sort_by_key(rmm::exec_policy(stream),
                             parent_col_id.begin(),
                             parent_col_id.begin() + num_list_parent,
                             scatter_indices.begin());

  rmm::device_uvector<size_type> row_offsets(num_nodes, stream, mr);
  // TODO is it possible to generate list child_offsets too here?
  // write only 1st child offset to parent node id child_offsets?
  thrust::exclusive_scan_by_key(rmm::exec_policy(stream),
                                parent_col_id.begin(),
                                parent_col_id.begin() + num_list_parent,
                                thrust::make_constant_iterator<size_type>(1),
                                row_offsets.begin());

  // Using scatter instead of sort.
  auto& temp_storage = parent_col_id;  // reuse parent_col_id as temp storage
  thrust::scatter(rmm::exec_policy(stream),
                  row_offsets.begin(),
                  row_offsets.begin() + num_list_parent,
                  scatter_indices.begin(),
                  temp_storage.begin());
  row_offsets = std::move(temp_storage);

  // Propagate row offsets to non-list leaves from list's immediate children node by recursion
  thrust::transform_if(
    rmm::exec_policy(stream),
    thrust::make_counting_iterator<size_type>(0),
    thrust::make_counting_iterator<size_type>(num_nodes),
    row_offsets.begin(),
    [node_categories = d_tree.node_categories.data(),
     parent_node_ids = d_tree.parent_node_ids.begin(),
     row_offsets     = row_offsets.begin()] __device__(size_type node_id) {
      auto parent_node_id = parent_node_ids[node_id];
      while (parent_node_id != parent_node_sentinel and
             node_categories[parent_node_id] != node_t::NC_LIST) {
        node_id        = parent_node_id;
        parent_node_id = parent_node_ids[parent_node_id];
      }
      return row_offsets[node_id];
    },
    [node_categories = d_tree.node_categories.data(),
     parent_node_ids = d_tree.parent_node_ids.begin()] __device__(size_type node_id) {
      auto const parent_node_id = parent_node_ids[node_id];
      return parent_node_id != parent_node_sentinel and
             !(node_categories[parent_node_id] == node_t::NC_LIST);
    });
  return row_offsets;
}

// This algorithm assigns a unique column id to each node in the tree.
// The row offset is the row index of the node in that column id.
// Algorithm:
// 1. Generate col_id:
//   a. Set operation on entire path of each node, translate each node id to set id.
//   b. gather unique set ids.
//   c. sort and use binary search to generate column ids.
//   d. Translate parent node ids to parent column ids.
// 2. Generate row_offset.
//   a. filter only list children
//   a. stable_sort by parent_col_id.
//   b. scan_by_key {parent_col_id} (done only on nodes whose parent is a list)
//   c. propagate to non-list leaves from parent list node by recursion
std::tuple<rmm::device_uvector<NodeIndexT>, rmm::device_uvector<size_type>>
records_orient_tree_traversal(device_span<SymbolT const> d_input,
                              tree_meta_t const& d_tree,
                              rmm::cuda_stream_view stream,
                              rmm::mr::device_memory_resource* mr)
{
  CUDF_FUNC_RANGE();
  auto [new_col_id, new_parent_col_id] = generate_column_id(d_input, d_tree, stream, mr);

  auto row_offsets = compute_row_offsets(std::move(new_parent_col_id), d_tree, stream, mr);
  return std::tuple{std::move(new_col_id), std::move(row_offsets)};
}

}  // namespace detail
}  // namespace cudf::io::json
