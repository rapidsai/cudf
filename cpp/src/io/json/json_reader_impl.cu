#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "json_reader_impl.hpp"

#include <hip/hip_runtime.h>

#include <algorithm>
#include <iostream>
#include <map>
#include <memory>
#include <numeric>
#include <string>
#include <vector>

#include <fcntl.h>
#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>

#include <thrust/device_ptr.h>
#include <thrust/execution_policy.h>

#include <thrust/host_vector.h>

#include <nvstrings/NVStrings.h>

#include <cudf/cudf.h>
#include <utilities/cudf_utils.h>
#include <utilities/error_utils.hpp>
#include <utilities/type_dispatcher.hpp>

#include <io/comp/io_uncomp.h>
#include <rmm/rmm.h>
#include <rmm/thrust_rmm_allocator.h>

#include <io/cuio_common.hpp>
#include <io/utilities/parsing_utils.cuh>
#include <io/utilities/wrapper_utils.hpp>

namespace cudf {
namespace io {
namespace json {

using string_pair = std::pair<const char *, size_t>;

reader::Impl::Impl(reader_options const &args) : args_(args) {
  // Check if the passed arguments are supported
  CUDF_EXPECTS(args_.lines, "Only Json Lines format is currently supported.\n");

  d_true_trie_ = createSerializedTrie({"true"});
  opts_.trueValuesTrie = d_true_trie_.data().get();

  d_false_trie_ = createSerializedTrie({"false"});
  opts_.falseValuesTrie = d_false_trie_.data().get();

  d_na_trie_ = createSerializedTrie({"null"});
  opts_.naValuesTrie = d_na_trie_.data().get();
}

/**---------------------------------------------------------------------------*
 * @brief Estimates the maximum expected length or a row, based on the number
 * of columns
 *
 * If the number of columns is not available, it will return a value large
 * enough for most use cases
 *
 * @param[in] num_columns Number of columns in the JSON file (optional)
 *
 * @return Estimated maximum size of a row, in bytes
 *---------------------------------------------------------------------------**/
constexpr size_t calculateMaxRowSize(int num_columns = 0) noexcept {
  constexpr size_t max_row_bytes = 16 * 1024; // 16KB
  constexpr size_t column_bytes = 64;
  constexpr size_t base_padding = 1024; // 1KB
  if (num_columns == 0) {
    // Use flat size if the number of columns is not known
    return max_row_bytes;
  } else {
    // Expand the size based on the number of columns, if available
    return base_padding + num_columns * column_bytes;
  }
}

table reader::Impl::read() {
  ingestRawInput();
  CUDF_EXPECTS(buffer_ != nullptr, "Ingest failed: input data is null.\n");

  decompressInput();
  CUDF_EXPECTS(uncomp_data_ != nullptr, "Ingest failed: uncompressed input data is null.\n");
  CUDF_EXPECTS(uncomp_size_ != 0, "Ingest failed: uncompressed input data has zero size.\n");

  setRecordStarts();
  CUDF_EXPECTS(!rec_starts_.empty(), "Error enumerating records.\n");

  uploadDataToDevice();
  CUDF_EXPECTS(!d_data_.empty(), "Error uploading input data to the GPU.\n");

  setColumnNames();
  CUDF_EXPECTS(!column_names_.empty(), "Error determining column names.\n");

  setDataTypes();
  CUDF_EXPECTS(!dtypes_.empty(), "Error in data type detection.\n");

  convertDataToColumns();
  CUDF_EXPECTS(!columns_.empty(), "Error converting json input into gdf columns.\n");

  // Transfer ownership to raw pointer output
  std::vector<gdf_column *> out_cols(columns_.size());
  for (size_t i = 0; i < columns_.size(); ++i) {
    out_cols[i] = columns_[i].release();
  }

  return table(out_cols.data(), out_cols.size());
}

table reader::Impl::read_byte_range(size_t offset, size_t size) {
  byte_range_offset_ = offset;
  byte_range_size_ = size;
  return read();
}

void reader::Impl::ingestRawInput() {
  size_t range_size = 0;
  if (byte_range_size_ != 0) {
    range_size = byte_range_size_ + calculateMaxRowSize(args_.dtype.size());
  }

  source_ = [&] {
    if (args_.source_type == FILE_PATH) {
      return datasource::create(args_.source, byte_range_offset_, range_size);
    } else if (args_.source_type == HOST_BUFFER) {
      return datasource::create(args_.source.c_str(), args_.source.size());
    } else {
      CUDF_FAIL("Invalid input type");
    }
  }();

  buffer_ = source_->get_buffer(byte_range_offset_,
                                std::max(byte_range_size_, source_->size()));
}

void reader::Impl::decompressInput() {
  const auto compression_type = inferCompressionType(
      args_.compression, args_.source_type, args_.source,
      {{"gz", "gzip"}, {"zip", "zip"}, {"bz2", "bz2"}, {"xz", "xz"}});
  if (compression_type == "none") {
    // Do not use the owner vector here to avoid extra copy
    uncomp_data_ = reinterpret_cast<const char *>(buffer_->data());
    uncomp_size_ = buffer_->size();
  } else {
    CUDF_EXPECTS(getUncompressedHostData(
                     reinterpret_cast<const char *>(buffer_->data()),
                     buffer_->size(), compression_type,
                     uncomp_data_owner_) == GDF_SUCCESS,
                 "Input data decompression failed.\n");
    uncomp_data_ = uncomp_data_owner_.data();
    uncomp_size_ = uncomp_data_owner_.size();
  }
}

void reader::Impl::setRecordStarts() {
  std::vector<char> chars_to_count{'\n'};
  // Currently, ignoring lineterminations within quotes is handled by recording the records of both,
  // and then filtering out the records that is a quotechar or a linetermination within a quotechar pair.
  if (allow_newlines_in_strings_) {
    chars_to_count.push_back('\"');
  }
  // If not starting at an offset, add an extra row to account for the first row in the file
  const auto prefilter_count =
      countAllFromSet(uncomp_data_, uncomp_size_, chars_to_count) + ((byte_range_offset_ == 0) ? 1 : 0);

  rec_starts_ = device_buffer<uint64_t>(prefilter_count);

  auto *find_result_ptr = rec_starts_.data();
  // Manually adding an extra row to account for the first row in the file
  if (byte_range_offset_ == 0) {
    find_result_ptr++;
    CUDA_TRY(hipMemsetAsync(rec_starts_.data(), 0ull, sizeof(uint64_t)));
  }

  std::vector<char> chars_to_find{'\n'};
  if (allow_newlines_in_strings_) {
    chars_to_find.push_back('\"');
  }
  // Passing offset = 1 to return positions AFTER the found character
  findAllFromSet(uncomp_data_, uncomp_size_, chars_to_find, 1, find_result_ptr);

  // Previous call stores the record pinput_file.typeositions as encountered by all threads
  // Sort the record positions as subsequent processing may require filtering
  // certain rows or other processing on specific records
  thrust::sort(rmm::exec_policy()->on(0), rec_starts_.data(), rec_starts_.data() + prefilter_count);

  auto filtered_count = prefilter_count;
  if (allow_newlines_in_strings_) {
    std::vector<uint64_t> h_rec_starts(prefilter_count);
    CUDA_TRY(
        hipMemcpy(h_rec_starts.data(), rec_starts_.data(), sizeof(uint64_t) * prefilter_count, hipMemcpyDefault));

    bool quotation = false;
    for (gdf_size_type i = 1; i < prefilter_count; ++i) {
      if (uncomp_data_[h_rec_starts[i] - 1] == '\"') {
        quotation = !quotation;
        h_rec_starts[i] = uncomp_size_;
        filtered_count--;
      } else if (quotation) {
        h_rec_starts[i] = uncomp_size_;
        filtered_count--;
      }
    }

    CUDA_TRY(hipMemcpy(rec_starts_.data(), h_rec_starts.data(), prefilter_count, hipMemcpyHostToDevice));
    thrust::sort(rmm::exec_policy()->on(0), rec_starts_.data(), rec_starts_.data() + prefilter_count);
  }

  // Exclude the ending newline as it does not precede a record start
  if (uncomp_data_[uncomp_size_ - 1] == '\n') {
    filtered_count--;
  }

  rec_starts_.resize(filtered_count);
}

void reader::Impl::uploadDataToDevice() {
  size_t start_offset = 0;
  size_t end_offset = uncomp_size_;

  // Trim lines that are outside range
  if (byte_range_size_ != 0 || byte_range_offset_ != 0) {
    std::vector<uint64_t> h_rec_starts(rec_starts_.size());
    CUDA_TRY(
        hipMemcpy(h_rec_starts.data(), rec_starts_.data(), sizeof(uint64_t) * h_rec_starts.size(), hipMemcpyDefault));

    if (byte_range_size_ != 0) {
      auto it = h_rec_starts.end() - 1;
      while (it >= h_rec_starts.begin() && *it > byte_range_size_) {
        end_offset = *it;
        --it;
      }
      h_rec_starts.erase(it + 1, h_rec_starts.end());
    }

    // Resize to exclude rows outside of the range; adjust row start positions to account for the data subcopy
    start_offset = h_rec_starts.front();
    rec_starts_.resize(h_rec_starts.size());
    thrust::transform(rmm::exec_policy()->on(0), rec_starts_.data(), rec_starts_.data() + rec_starts_.size(),
                      thrust::make_constant_iterator(start_offset), rec_starts_.data(), thrust::minus<uint64_t>());
  }

  const size_t bytes_to_upload = end_offset - start_offset;
  CUDF_EXPECTS(bytes_to_upload <= uncomp_size_, "Error finding the record within the specified byte range.\n");

  // Upload the raw data that is within the rows of interest
  d_data_ = device_buffer<char>(bytes_to_upload);
  CUDA_TRY(hipMemcpy(d_data_.data(), uncomp_data_ + start_offset, bytes_to_upload, hipMemcpyHostToDevice));
}

/**---------------------------------------------------------------------------*
 * @brief Extract value names from a JSON object
 *
 * @param[in] json_obj Host vector containing the JSON object
 * @param[in] opts Parsing options (e.g. delimiter and quotation character)
 *
 * @return std::vector<std::string> names of JSON object values
 *---------------------------------------------------------------------------**/
std::vector<std::string> getNamesFromJsonObject(const std::vector<char> &json_obj, const ParseOptions &opts) {
  enum class ParseState { preColName, colName, postColName };
  std::vector<std::string> names;
  bool quotation = false;
  auto state = ParseState::preColName;
  int name_start = 0;
  for (size_t pos = 0; pos < json_obj.size(); ++pos) {
    if (state == ParseState::preColName) {
      if (json_obj[pos] == opts.quotechar) {
        name_start = pos + 1;
        state = ParseState::colName;
        continue;
      }
    } else if (state == ParseState::colName) {
      if (json_obj[pos] == opts.quotechar && json_obj[pos - 1] != '\\') {
        // if found a non-escaped quote character, it's the end of the column name
        names.emplace_back(&json_obj[name_start], &json_obj[pos]);
        state = ParseState::postColName;
        continue;
      }
    } else if (state == ParseState::postColName) {
      // TODO handle complex data types that might include unquoted commas
      if (!quotation && json_obj[pos] == opts.delimiter) {
        state = ParseState::preColName;
        continue;
      } else if (json_obj[pos] == opts.quotechar) {
        quotation = !quotation;
      }
    }
  }
  return names;
}

void reader::Impl::setColumnNames() {
  // If file only contains one row, use the file size for the row size
  uint64_t first_row_len = d_data_.size() / sizeof(char);
  if (rec_starts_.size() > 1) {
    // Set first_row_len to the offset of the second row, if it exists
    CUDA_TRY(hipMemcpy(&first_row_len, rec_starts_.data() + 1, sizeof(uint64_t), hipMemcpyDefault));
  }
  std::vector<char> first_row(first_row_len);
  CUDA_TRY(hipMemcpy(first_row.data(), d_data_.data(), first_row_len * sizeof(char), hipMemcpyDefault));

  // Determine the row format between:
  //   JSON array - [val1, val2, ...] and
  //   JSON object - {"col1":val1, "col2":val2, ...}
  // based on the top level opening bracket
  const auto first_square_bracket = std::find(first_row.begin(), first_row.end(), '[');
  const auto first_curly_bracket = std::find(first_row.begin(), first_row.end(), '{');
  CUDF_EXPECTS(first_curly_bracket != first_row.end() || first_square_bracket != first_row.end(),
               "Input data is not a valid JSON file.");
  // If the first opening bracket is '{', assume object format
  const bool is_object = first_curly_bracket < first_square_bracket;
  if (is_object) {
    column_names_ = getNamesFromJsonObject(first_row, opts_);
  } else {
    int cols_found = 0;
    bool quotation = false;
    for (size_t pos = 0; pos < first_row.size(); ++pos) {
      // Flip the quotation flag if current character is a quotechar
      if (first_row[pos] == opts_.quotechar) {
        quotation = !quotation;
      }
      // Check if end of a column/row
      else if (pos == first_row.size() - 1 || (!quotation && first_row[pos] == opts_.delimiter)) {
        column_names_.emplace_back(std::to_string(cols_found++));
      }
    }
  }
}

void reader::Impl::convertDataToColumns() {
  const auto num_columns = dtypes_.size();

  for (size_t col = 0; col < num_columns; ++col) {
    columns_.emplace_back(rec_starts_.size(), dtypes_[col], gdf_dtype_extra_info{TIME_UNIT_NONE}, column_names_[col]);
    CUDF_EXPECTS(columns_.back().allocate() == GDF_SUCCESS, "Cannot allocate columns.\n");
  }

  thrust::host_vector<gdf_dtype> h_dtypes(num_columns);
  thrust::host_vector<void *> h_data(num_columns);
  thrust::host_vector<gdf_valid_type *> h_valid(num_columns);

  for (size_t i = 0; i < num_columns; ++i) {
    h_dtypes[i] = columns_[i]->dtype;
    h_data[i] = columns_[i]->data;
    h_valid[i] = columns_[i]->valid;
  }

  rmm::device_vector<gdf_dtype> d_dtypes = h_dtypes;
  rmm::device_vector<void *> d_data = h_data;
  rmm::device_vector<gdf_valid_type *> d_valid = h_valid;
  rmm::device_vector<gdf_size_type> d_valid_counts(num_columns, 0);

  convertJsonToColumns(d_dtypes.data().get(), d_data.data().get(), d_valid.data().get(), d_valid_counts.data().get());
  CUDA_TRY(hipDeviceSynchronize());
  CUDA_TRY(hipGetLastError());

  thrust::host_vector<gdf_size_type> h_valid_counts = d_valid_counts;
  for (size_t i = 0; i < num_columns; ++i) {
    columns_[i]->null_count = columns_[i]->size - h_valid_counts[i];
  }

  // Handle string columns
  for (auto &column : columns_) {
    if (column->dtype == GDF_STRING) {
      auto str_list = static_cast<string_pair *>(column->data);
      auto str_data = NVStrings::create_from_index(str_list, column->size);
      RMM_FREE(std::exchange(column->data, str_data), 0);
    }
  }
}

/**---------------------------------------------------------------------------*
 * @brief Functor for converting plain text data to cuDF data type value.
 *---------------------------------------------------------------------------**/
struct ConvertFunctor {
  /**---------------------------------------------------------------------------*
   * @brief Template specialization for operator() for types whose values can be
   * convertible to a 0 or 1 to represent false/true. The converting is done by
   * checking against the default and user-specified true/false values list.
   *
   * It is handled here rather than within convertStrToValue() as that function
   * is used by other types (ex. timestamp) that aren't 'booleable'.
   *---------------------------------------------------------------------------**/
  template <typename T, typename std::enable_if_t<std::is_integral<T>::value> * = nullptr>
  __host__ __device__ __forceinline__ void operator()(const char *data, void *gdf_columns, long row, long start,
                                                      long end, const ParseOptions &opts) {
    T &value{static_cast<T *>(gdf_columns)[row]};

    // Check for user-specified true/false values first, where the output is
    // replaced with 1/0 respectively
    const size_t field_len = end - start + 1;
    if (serializedTrieContains(opts.trueValuesTrie, data + start, field_len)) {
      value = 1;
    } else if (serializedTrieContains(opts.falseValuesTrie, data + start, field_len)) {
      value = 0;
    } else {
      value = convertStrToValue<T>(data, start, end, opts);
    }
  }

  /**---------------------------------------------------------------------------*
   * @brief Default template operator() dispatch specialization all data types
   * (including wrapper types) that is not covered by above.
   *---------------------------------------------------------------------------**/
  template <typename T, typename std::enable_if_t<!std::is_integral<T>::value> * = nullptr>
  __host__ __device__ __forceinline__ void operator()(const char *data, void *gdf_columns, long row, long start,
                                                      long end, const ParseOptions &opts) {
    T &value{static_cast<T *>(gdf_columns)[row]};
    value = convertStrToValue<T>(data, start, end, opts);
  }
};

/**---------------------------------------------------------------------------*
 * @brief CUDA Kernel that modifies the start and stop offsets to exclude
 * the sections outside of the top level brackets.
 *
 * The top level brackets characters are excluded from the resulting range.
 * Parameter stop has the same semantics as end() in STL containers
 * (one past the last element)
 *
 * @param[in] data Pointer to the device buffer containing the data to process
 * @param[in,out] start Offset of the first character in the range
 * @param[in,out] stop Offset of the first character after the range
 *
 * @return void
 *---------------------------------------------------------------------------**/
__device__ void limitRangeToBrackets(const char *data, long &start, long &stop) {
  while (start < stop && data[start] != '[' && data[start] != '{') {
    start++;
  }
  start++;

  while (start < stop && data[stop - 1] != ']' && data[stop - 1] != '}') {
    stop--;
  }
  stop--;
}

/**---------------------------------------------------------------------------*
 * @brief CUDA kernel that finds the end position of the next field name,
 * including the colon that separates the name from the field value.
 *
 * Returns the position after the colon that preceeds the value token.
 *
 * @param[in] data Pointer to the device buffer containing the data to process
 * @param[in] opts Parsing options (e.g. delimiter and quotation character)
 * @param[in] start Offset of the first character in the range
 * @param[in] stop Offset of the first character after the range
 *
 * @return long Position of the first character after the field name.
 *---------------------------------------------------------------------------**/
__device__ long seekFieldNameEnd(const char *data, const ParseOptions opts, long start, long stop) {
  bool quotation = false;
  for (auto pos = start; pos < stop; ++pos) {
    // Ignore escaped quotes
    if (data[pos] == opts.quotechar && data[pos - 1] != '\\') {
      quotation = !quotation;
    } else if (!quotation && data[pos] == ':') {
      return pos + 1;
    }
  }
  return stop;
}

/**---------------------------------------------------------------------------*
 * @brief CUDA kernel that parses and converts plain text data into cuDF column data.
 *
 * Data is processed one record at a time
 *
 * @param[in] data The entire data to read
 * @param[in] data_size Size of the data buffer, in bytes
 * @param[in] rec_starts The start of each data record
 * @param[in] num_records The number of lines/rows
 * @param[in] dtypes The data type of each column
 * @param[in] opts A set of parsing options
 * @param[out] gdf_columns The output column data
 * @param[in] num_columns The number of columns
 * @param[out] valid_fields The bitmaps indicating whether column fields are valid
 * @param[out] num_valid_fields The numbers of valid fields in columns
 *
 * @return void
 *---------------------------------------------------------------------------**/
__global__ void convertJsonToGdf(const char *data, size_t data_size, const uint64_t *rec_starts,
                                 gdf_size_type num_records, const gdf_dtype *dtypes, ParseOptions opts,
                                 void *const *gdf_columns, int num_columns, gdf_valid_type *const *valid_fields,
                                 gdf_size_type *num_valid_fields) {
  const long rec_id = threadIdx.x + (blockDim.x * blockIdx.x);
  if (rec_id >= num_records)
    return;

  long start = rec_starts[rec_id];
  // has the same semantics as end() in STL containers (one past last element)
  long stop = ((rec_id < num_records - 1) ? rec_starts[rec_id + 1] : data_size);

  limitRangeToBrackets(data, start, stop);
  const bool is_object = (data[start - 1] == '{');

  for (int col = 0; col < num_columns && start < stop; col++) {
    if (is_object) {
      start = seekFieldNameEnd(data, opts, start, stop);
    }
    // field_end is at the next delimiter/newline
    const long field_end = seekFieldEnd(data, opts, start, stop);
    long field_data_last = field_end - 1;
    // Modify start & end to ignore whitespace and quotechars
    adjustForWhitespaceAndQuotes(data, &start, &field_data_last, opts.quotechar);
    // Empty fields are not legal values
    if (start <= field_data_last && !serializedTrieContains(opts.naValuesTrie, data + start, field_end - start)) {
      // Type dispatcher does not handle GDF_STRINGS
      if (dtypes[col] == gdf_dtype::GDF_STRING) {
        auto str_list = static_cast<string_pair *>(gdf_columns[col]);
        str_list[rec_id].first = data + start;
        str_list[rec_id].second = field_data_last - start + 1;
      } else {
        cudf::type_dispatcher(dtypes[col], ConvertFunctor{}, data, gdf_columns[col], rec_id, start, field_data_last,
                              opts);
      }

      // set the valid bitmap - all bits were set to 0 to start
      setBitmapBit(valid_fields[col], rec_id);
      atomicAdd(&num_valid_fields[col], 1);
    } else if (dtypes[col] == gdf_dtype::GDF_STRING) {
      auto str_list = static_cast<string_pair *>(gdf_columns[col]);
      str_list[rec_id].first = nullptr;
      str_list[rec_id].second = 0;
    }
    start = field_end + 1;
  }
}

void reader::Impl::convertJsonToColumns(gdf_dtype *const dtypes, void *const *gdf_columns,
                                            gdf_valid_type *const *valid_fields, gdf_size_type *num_valid_fields) {
  int block_size;
  int min_grid_size;
  CUDA_TRY(hipOccupancyMaxPotentialBlockSize(&min_grid_size, &block_size, convertJsonToGdf));

  const int grid_size = (rec_starts_.size() + block_size - 1) / block_size;

  convertJsonToGdf<<<grid_size, block_size>>>(d_data_.data(), d_data_.size(), rec_starts_.data(), rec_starts_.size(),
                                              dtypes, opts_, gdf_columns, columns_.size(), valid_fields,
                                              num_valid_fields);

  CUDA_TRY(hipGetLastError());
}

/**---------------------------------------------------------------------------*
 * @brief CUDA kernel that parses and converts data into cuDF column data.
 *
 * Data is processed in one row/record at a time, so the number of total
 * threads (tid) is equal to the number of rows.
 *
 * @param[in] data The entire plain text data to read
 * @param[in] data_size Size of the data buffer, in bytes
 * @param[in] opts A set of parsing options
 * @param[in] num_columns The number of columns of input data
 * @param[in] rec_starts The start the input data of interest
 * @param[in] num_records The number of lines/rows of input data
 * @param[out] column_infos The count for each column data type
 *
 * @returns void
 *---------------------------------------------------------------------------**/
__global__ void detectJsonDataTypes(const char *data, size_t data_size, const ParseOptions opts, int num_columns,
                                    const uint64_t *rec_starts, gdf_size_type num_records, ColumnInfo *column_infos) {
  long rec_id = threadIdx.x + (blockDim.x * blockIdx.x);
  if (rec_id >= num_records)
    return;

  long start = rec_starts[rec_id];
  // has the same semantics as end() in STL containers (one past last element)
  long stop = ((rec_id < num_records - 1) ? rec_starts[rec_id + 1] : data_size);

  limitRangeToBrackets(data, start, stop);
  const bool is_object = (data[start - 1] == '{');

  for (int col = 0; col < num_columns; col++) {
    if (is_object) {
      start = seekFieldNameEnd(data, opts, start, stop);
    }
    const long field_end = seekFieldEnd(data, opts, start, stop);
    long field_data_last = field_end - 1;
    adjustForWhitespaceAndQuotes(data, &start, &field_data_last);
    const int field_len = field_data_last - start + 1;

    // Checking if the field is empty
    if (start > field_data_last || serializedTrieContains(opts.naValuesTrie, data + start, field_len)) {
      atomicAdd(&column_infos[col].null_count, 1);
      start = field_end + 1;
      continue;
    }

    int digit_count = 0;
    int decimal_count = 0;
    int slash_count = 0;
    int dash_count = 0;
    int colon_count = 0;
    int exponent_count = 0;
    int other_count = 0;

    const bool maybe_hex = ((field_len > 2 && data[start] == '0' && data[start + 1] == 'x') ||
                            (field_len > 3 && data[start] == '-' && data[start + 1] == '0' && data[start + 2] == 'x'));
    for (long pos = start; pos <= field_data_last; pos++) {
      if (isDigit(data[pos], maybe_hex)) {
        digit_count++;
        continue;
      }
      // Looking for unique characters that will help identify column types
      switch (data[pos]) {
      case '.':
        decimal_count++;
        break;
      case '-':
        dash_count++;
        break;
      case '/':
        slash_count++;
        break;
      case ':':
        colon_count++;
        break;
      case 'e':
      case 'E':
        if (!maybe_hex && pos > start && pos < field_data_last)
          exponent_count++;
        break;
      default:
        other_count++;
        break;
      }
    }

    // Integers have to have the length of the string
    int int_req_number_cnt = field_len;
    // Off by one if they start with a minus sign
    if (data[start] == '-' && field_len > 1) {
      --int_req_number_cnt;
    }
    // Off by one if they are a hexadecimal number
    if (maybe_hex) {
      --int_req_number_cnt;
    }
    if (serializedTrieContains(opts.trueValuesTrie, data + start, field_len) ||
        serializedTrieContains(opts.falseValuesTrie, data + start, field_len)) {
      atomicAdd(&column_infos[col].bool_count, 1);
    } else if (digit_count == int_req_number_cnt) {
      atomicAdd(&column_infos[col].int_count, 1);
    } else if (isLikeFloat(field_len, digit_count, decimal_count, dash_count, exponent_count)) {
      atomicAdd(&column_infos[col].float_count, 1);
    }
    // A date-time field cannot have more than 3 non-special characters
    // A number field cannot have more than one decimal point
    else if (other_count > 3 || decimal_count > 1) {
      atomicAdd(&column_infos[col].string_count, 1);
    } else {
      // A date field can have either one or two '-' or '\'; A legal combination will only have one of them
      // To simplify the process of auto column detection, we are not covering all the date-time formation permutations
      if ((dash_count > 0 && dash_count <= 2 && slash_count == 0) ||
          (dash_count == 0 && slash_count > 0 && slash_count <= 2)) {
        if (colon_count <= 2) {
          atomicAdd(&column_infos[col].datetime_count, 1);
        } else {
          atomicAdd(&column_infos[col].string_count, 1);
        }
      } else {
        // Default field type is string
        atomicAdd(&column_infos[col].string_count, 1);
      }
    }
    start = field_end + 1;
  }
}

void reader::Impl::detectDataTypes(ColumnInfo *column_infos) {
  int block_size;
  int min_grid_size;
  CUDA_TRY(hipOccupancyMaxPotentialBlockSize(&min_grid_size, &block_size, detectJsonDataTypes));

  // Calculate actual block count to use based on records count
  const int grid_size = (rec_starts_.size() + block_size - 1) / block_size;

  detectJsonDataTypes<<<grid_size, block_size>>>(d_data_.data(), d_data_.size(), opts_, column_names_.size(),
                                                 rec_starts_.data(), rec_starts_.size(), column_infos);

  CUDA_TRY(hipGetLastError());
}

void reader::Impl::setDataTypes() {
  if (!args_.dtype.empty()) {
    CUDF_EXPECTS(args_.dtype.size() == column_names_.size(), "Need to specify the type of each column.\n");
    // Assume that the dtype is in dictionary format only if all elements contain a colon
    const bool is_dict = std::all_of(args_.dtype.begin(), args_.dtype.end(), [](const std::string &s) {
      return std::find(s.begin(), s.end(), ':') != s.end();
    });
    if (is_dict) {
      std::map<std::string, gdf_dtype> col_type_map;
      for (const auto &ts : args_.dtype) {
        const size_t colon_idx = ts.find(":");
        const std::string col_name(ts.begin(), ts.begin() + colon_idx);
        const std::string type_str(ts.begin() + colon_idx + 1, ts.end());
        col_type_map[col_name] = convertStringToDtype(type_str);
      }

      // Using the map here allows O(n log n) complexity
      for (size_t col = 0; col < args_.dtype.size(); ++col) {
        dtypes_.push_back(col_type_map[column_names_[col]]);
      }
    } else {
      for (size_t col = 0; col < args_.dtype.size(); ++col) {
        dtypes_.push_back(convertStringToDtype(args_.dtype[col]));
      }
    }
  } else {
    CUDF_EXPECTS(rec_starts_.size() != 0, "No data available for data type inference.\n");
    const auto num_columns = column_names_.size();

    rmm::device_vector<ColumnInfo> d_column_infos(num_columns, ColumnInfo{});
    detectDataTypes(d_column_infos.data().get());
    thrust::host_vector<ColumnInfo> h_column_infos = d_column_infos;

    for (const auto &cinfo : h_column_infos) {
      if (cinfo.null_count == static_cast<int>(rec_starts_.size())) {
        // Entire column is NULL; allocate the smallest amount of memory
        dtypes_.push_back(GDF_INT8);
      } else if (cinfo.string_count > 0) {
        dtypes_.push_back(GDF_STRING);
      } else if (cinfo.datetime_count > 0) {
        dtypes_.push_back(GDF_DATE64);
      } else if (cinfo.float_count > 0 || (cinfo.int_count > 0 && cinfo.null_count > 0)) {
        dtypes_.push_back(GDF_FLOAT64);
      } else if (cinfo.int_count > 0) {
        dtypes_.push_back(GDF_INT64);
      } else if (cinfo.bool_count > 0) {
        dtypes_.push_back(GDF_BOOL8);
      } else {
        CUDF_FAIL("Data type detection failed.\n");
      }
    }
  }
}

reader::reader(reader_options const &args)
    : impl_(std::make_unique<Impl>(args)) {}

table reader::read() { return impl_->read(); }

table reader::read_byte_range(size_t offset, size_t size) {
  return impl_->read_byte_range(offset, size);
}

reader::~reader() = default;

} // namespace json
} // namespace io
} // namespace cudf
