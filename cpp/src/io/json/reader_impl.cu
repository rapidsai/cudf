#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2020, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

/**
 * @file reader_impl.cu
 * @brief cuDF-IO JSON reader class implementation
 */

#include "reader_impl.hpp"

#include <io/comp/io_uncomp.h>
#include <io/utilities/parsing_utils.cuh>
#include <io/utilities/type_conversion.cuh>

#include <cudf/column/column_factories.hpp>
#include <cudf/detail/utilities/trie.cuh>
#include <cudf/groupby.hpp>
#include <cudf/sorting.hpp>
#include <cudf/strings/detail/replace.hpp>
#include <cudf/table/table.hpp>
#include <cudf/utilities/error.hpp>
#include <cudf/utilities/span.hpp>

#include <rmm/thrust_rmm_allocator.h>
#include <rmm/device_scalar.hpp>

#include <thrust/optional.h>

using cudf::detail::host_span;

namespace cudf {
namespace io {
namespace detail {
namespace json {
using namespace cudf::io;

namespace {

/**
 * @brief Estimates the maximum expected length or a row, based on the number
 * of columns
 *
 * If the number of columns is not available, it will return a value large
 * enough for most use cases
 *
 * @param[in] num_columns Number of columns in the JSON file (optional)
 *
 * @return Estimated maximum size of a row, in bytes
 */
constexpr size_t calculate_max_row_size(int num_columns = 0) noexcept
{
  constexpr size_t max_row_bytes = 16 * 1024;  // 16KB
  constexpr size_t column_bytes  = 64;
  constexpr size_t base_padding  = 1024;  // 1KB
  return num_columns == 0
           ? max_row_bytes  // Use flat size if the # of columns is not known
           : base_padding +
               num_columns * column_bytes;  // Expand size based on the # of columns, if available
}

}  // anonymous namespace

/**
 * @brief Aggregate the table containing keys info by their hash values.
 *
 * @param[in] info Table with columns containing key offsets, lengths and hashes, respectively
 *
 * @return Table with data aggregated by key hash values
 */
std::unique_ptr<table> aggregate_keys_info(std::unique_ptr<table> info)
{
  auto const info_view = info->view();
  std::vector<groupby::aggregation_request> requests;
  requests.emplace_back(groupby::aggregation_request{info_view.column(0)});
  requests.back().aggregations.emplace_back(make_min_aggregation());
  requests.back().aggregations.emplace_back(make_nth_element_aggregation(0));

  requests.emplace_back(groupby::aggregation_request{info_view.column(1)});
  requests.back().aggregations.emplace_back(make_min_aggregation());
  requests.back().aggregations.emplace_back(make_nth_element_aggregation(0));

  // Aggregate by hash values
  groupby::groupby gb_obj(
    table_view({info_view.column(2)}), null_policy::EXCLUDE, sorted::NO, {}, {});

  auto result = gb_obj.aggregate(requests);  // TODO: no stream parameter?

  std::vector<std::unique_ptr<column>> out_columns;
  out_columns.emplace_back(std::move(result.second[0].results[0]));  // offsets
  out_columns.emplace_back(std::move(result.second[1].results[0]));  // lengths
  out_columns.emplace_back(std::move(result.first->release()[0]));   // hashes
  return std::make_unique<table>(std::move(out_columns));
}

/**
 * @brief Initializes the (key hash -> column index) hash map.
 */
col_map_ptr_type create_col_names_hash_map(column_view column_name_hashes, hipStream_t stream)
{
  auto key_col_map{col_map_type::create(column_name_hashes.size())};
  auto const column_data = column_name_hashes.data<uint32_t>();
  thrust::for_each_n(rmm::exec_policy(stream)->on(stream),
                     thrust::make_counting_iterator<size_type>(0),
                     column_name_hashes.size(),
                     [map = *key_col_map, column_data] __device__(size_type idx) mutable {
                       map.insert(thrust::make_pair(column_data[idx], idx));
                     });
  return key_col_map;
}

/**
 * @brief Create a table whose columns contain the information on JSON objects' keys.
 *
 * The columns contain name offsets in the file, name lengths and name hashes, respectively.
 *
 * @param[in] options Parsing options (e.g. delimiter and quotation character)
 * @param[in] data Input JSON device data
 * @param[in] row_offsets Device array of row start locations in the input buffer
 * @param[in] stream CUDA stream used for device memory operations and kernel launches
 *
 * @return std::unique_ptr<table> cudf table with three columns (offsets, lenghts, hashes)
 */
std::unique_ptr<table> create_json_keys_info_table(const parse_options_view &options,
                                                   device_span<char const> const data,
                                                   device_span<uint64_t const> const row_offsets,
                                                   hipStream_t stream)
{
  // Count keys
  rmm::device_scalar<unsigned long long int> key_counter(0, stream);
  cudf::io::json::gpu::collect_keys_info(
    options, data, row_offsets, key_counter.data(), {}, stream);

  // Allocate columns to store hash value, length, and offset of each JSON object key in the input
  auto const num_keys = key_counter.value();
  std::vector<std::unique_ptr<column>> info_columns;
  info_columns.emplace_back(make_numeric_column(data_type(type_id::UINT64), num_keys));
  info_columns.emplace_back(make_numeric_column(data_type(type_id::UINT16), num_keys));
  info_columns.emplace_back(make_numeric_column(data_type(type_id::UINT32), num_keys));
  // Create a table out of these columns to pass them around more easily
  auto info_table           = std::make_unique<table>(std::move(info_columns));
  auto const info_table_mdv = mutable_table_device_view::create(info_table->mutable_view(), stream);

  // Reset the key counter - now used for indexing
  key_counter.set_value(0, stream);
  // Fill the allocated columns
  cudf::io::json::gpu::collect_keys_info(
    options, data, row_offsets, key_counter.data(), {*info_table_mdv}, stream);
  return info_table;
}

/**
 * @brief Extract the keys from the JSON file the name offsets/lengths.
 */
std::vector<std::string> create_key_strings(char const *h_data,
                                            table_view sorted_info,
                                            hipStream_t stream)
{
  auto const num_cols = sorted_info.num_rows();
  std::vector<uint64_t> h_offsets(num_cols);
  hipMemcpyAsync(h_offsets.data(),
                  sorted_info.column(0).data<uint64_t>(),
                  sizeof(uint64_t) * num_cols,
                  hipMemcpyDefault,
                  stream);

  std::vector<uint16_t> h_lens(num_cols);
  hipMemcpyAsync(h_lens.data(),
                  sorted_info.column(1).data<uint16_t>(),
                  sizeof(uint16_t) * num_cols,
                  hipMemcpyDefault,
                  stream);

  std::vector<std::string> names(num_cols);
  std::transform(h_offsets.cbegin(),
                 h_offsets.cend(),
                 h_lens.cbegin(),
                 names.begin(),
                 [&](auto offset, auto len) { return std::string(h_data + offset, len); });
  return names;
}

auto sort_keys_info_by_offset(std::unique_ptr<table> info)
{
  auto const agg_offset_col_view = info->get_column(0).view();
  return sort_by_key(info->view(), table_view({agg_offset_col_view}));
}

/**
 * @brief Extract JSON object keys from a JSON file.
 *
 * @param[in] stream CUDA stream used for device memory operations and kernel launches.
 *
 * @return Names of JSON object keys in the file
 */
std::pair<std::vector<std::string>, col_map_ptr_type> reader::impl::get_json_object_keys_hashes(
  hipStream_t stream)
{
  auto info = create_json_keys_info_table(
    opts_.view(),
    device_span<char const>(static_cast<char const *>(data_.data()), data_.size()),
    rec_starts_,
    stream);

  auto aggregated_info = aggregate_keys_info(std::move(info));
  auto sorted_info     = sort_keys_info_by_offset(std::move(aggregated_info));

  return {create_key_strings(uncomp_data_, sorted_info->view(), stream),
          create_col_names_hash_map(sorted_info->get_column(2).view(), stream)};
}

/**
 * @brief Ingest input JSON file/buffer, without decompression.
 *
 * Sets the source_, byte_range_offset_, and byte_range_size_ data members
 *
 * @param[in] range_offset Number of bytes offset from the start
 * @param[in] range_size Bytes to read; use `0` for all remaining data
 */
void reader::impl::ingest_raw_input(size_t range_offset, size_t range_size)
{
  size_t map_range_size = 0;
  if (range_size != 0) {
    map_range_size = range_size + calculate_max_row_size(options_.get_dtypes().size());
  }

  // Support delayed opening of the file if using memory mapping datasource
  // This allows only mapping of a subset of the file if using byte range
  if (source_ == nullptr) {
    assert(!filepath_.empty());
    source_ = datasource::create(filepath_, range_offset, map_range_size);
  }

  if (!source_->is_empty()) {
    auto data_size = (map_range_size != 0) ? map_range_size : source_->size();
    buffer_        = source_->host_read(range_offset, data_size);
  }

  byte_range_offset_ = range_offset;
  byte_range_size_   = range_size;
  load_whole_file_   = byte_range_offset_ == 0 && byte_range_size_ == 0;
}

/**
 * @brief Decompress the input data, if needed
 *
 * Sets the uncomp_data_ and uncomp_size_ data members
 * Loads the data into device memory if byte range parameters are not used
 */
void reader::impl::decompress_input(hipStream_t stream)
{
  const auto compression_type =
    infer_compression_type(options_.get_compression(),
                           filepath_,
                           {{"gz", "gzip"}, {"zip", "zip"}, {"bz2", "bz2"}, {"xz", "xz"}});
  if (compression_type == "none") {
    // Do not use the owner vector here to avoid extra copy
    uncomp_data_ = reinterpret_cast<const char *>(buffer_->data());
    uncomp_size_ = buffer_->size();
  } else {
    uncomp_data_owner_ = get_uncompressed_data(  //
      host_span<char const>(                     //
        reinterpret_cast<const char *>(buffer_->data()),
        buffer_->size()),
      compression_type);

    uncomp_data_ = uncomp_data_owner_.data();
    uncomp_size_ = uncomp_data_owner_.size();
  }
  if (load_whole_file_) data_ = rmm::device_buffer(uncomp_data_, uncomp_size_, stream);
}

/**
 * @brief Finds all record starts in the file and stores them in rec_starts_
 *
 * Does not upload the entire file to the GPU
 *
 * @param[in] stream CUDA stream used for device memory operations and kernel launches.
 */
void reader::impl::set_record_starts(hipStream_t stream)
{
  std::vector<char> chars_to_count{'\n'};
  // Currently, ignoring lineterminations within quotes is handled by recording the records of both,
  // and then filtering out the records that is a quotechar or a linetermination within a quotechar
  // pair.
  if (allow_newlines_in_strings_) { chars_to_count.push_back('\"'); }
  // If not starting at an offset, add an extra row to account for the first row in the file
  cudf::size_type prefilter_count = ((byte_range_offset_ == 0) ? 1 : 0);
  if (load_whole_file_) {
    prefilter_count += count_all_from_set(data_, chars_to_count);
  } else {
    prefilter_count += count_all_from_set(uncomp_data_, uncomp_size_, chars_to_count);
  }

  rec_starts_.resize(prefilter_count);

  auto *find_result_ptr = rec_starts_.data().get();
  // Manually adding an extra row to account for the first row in the file
  if (byte_range_offset_ == 0) {
    find_result_ptr++;
    CUDA_TRY(hipMemsetAsync(rec_starts_.data().get(), 0ull, sizeof(uint64_t), stream));
  }

  std::vector<char> chars_to_find{'\n'};
  if (allow_newlines_in_strings_) { chars_to_find.push_back('\"'); }
  // Passing offset = 1 to return positions AFTER the found character
  if (load_whole_file_) {
    find_all_from_set(data_, chars_to_find, 1, find_result_ptr);
  } else {
    find_all_from_set(uncomp_data_, uncomp_size_, chars_to_find, 1, find_result_ptr);
  }

  // Previous call stores the record pinput_file.typeositions as encountered by all threads
  // Sort the record positions as subsequent processing may require filtering
  // certain rows or other processing on specific records
  thrust::sort(rmm::exec_policy()->on(stream), rec_starts_.begin(), rec_starts_.end());

  auto filtered_count = prefilter_count;
  if (allow_newlines_in_strings_) {
    thrust::host_vector<uint64_t> h_rec_starts = rec_starts_;
    bool quotation                             = false;
    for (cudf::size_type i = 1; i < prefilter_count; ++i) {
      if (uncomp_data_[h_rec_starts[i] - 1] == '\"') {
        quotation       = !quotation;
        h_rec_starts[i] = uncomp_size_;
        filtered_count--;
      } else if (quotation) {
        h_rec_starts[i] = uncomp_size_;
        filtered_count--;
      }
    }

    rec_starts_ = h_rec_starts;
    thrust::sort(rmm::exec_policy()->on(stream), rec_starts_.begin(), rec_starts_.end());
  }

  // Exclude the ending newline as it does not precede a record start
  if (uncomp_data_[uncomp_size_ - 1] == '\n') { filtered_count--; }

  rec_starts_.resize(filtered_count);
}

/**
 * @brief Uploads the relevant segment of the input json data onto the GPU.
 *
 * Sets the d_data_ data member.
 * Only rows that need to be parsed are copied, based on the byte range
 * Also updates the array of record starts to match the device data offset.
 *
 */
void reader::impl::upload_data_to_device(hipStream_t stream)
{
  size_t start_offset = 0;
  size_t end_offset   = uncomp_size_;

  // Trim lines that are outside range
  if (byte_range_size_ != 0 || byte_range_offset_ != 0) {
    thrust::host_vector<uint64_t> h_rec_starts = rec_starts_;

    if (byte_range_size_ != 0) {
      auto it = h_rec_starts.end() - 1;
      while (it >= h_rec_starts.begin() && *it > byte_range_size_) {
        end_offset = *it;
        --it;
      }
      h_rec_starts.erase(it + 1, h_rec_starts.end());
    }

    // Resize to exclude rows outside of the range
    // Adjust row start positions to account for the data subcopy
    start_offset = h_rec_starts.front();
    rec_starts_.resize(h_rec_starts.size());
    thrust::transform(rmm::exec_policy()->on(stream),
                      rec_starts_.begin(),
                      rec_starts_.end(),
                      thrust::make_constant_iterator(start_offset),
                      rec_starts_.begin(),
                      thrust::minus<uint64_t>());
  }

  const size_t bytes_to_upload = end_offset - start_offset;
  CUDF_EXPECTS(bytes_to_upload <= uncomp_size_,
               "Error finding the record within the specified byte range.\n");

  // Upload the raw data that is within the rows of interest
  data_ = rmm::device_buffer(uncomp_data_ + start_offset, bytes_to_upload, stream);
}

/**
 * @brief Parse the first row to set the column name
 *
 * Sets the column_names_ data member
 *
 * @param[in] stream CUDA stream used for device memory operations and kernel launches.
 */
void reader::impl::set_column_names(hipStream_t stream)
{
  // If file only contains one row, use the file size for the row size
  uint64_t first_row_len = data_.size() / sizeof(char);
  if (rec_starts_.size() > 1) {
    // Set first_row_len to the offset of the second row, if it exists
    CUDA_TRY(hipMemcpyAsync(&first_row_len,
                             rec_starts_.data().get() + 1,
                             sizeof(uint64_t),
                             hipMemcpyDeviceToHost,
                             stream));
  }
  std::vector<char> first_row(first_row_len);
  CUDA_TRY(hipMemcpyAsync(
    first_row.data(), data_.data(), first_row_len * sizeof(char), hipMemcpyDeviceToHost, stream));
  CUDA_TRY(hipStreamSynchronize(stream));

  // Determine the row format between:
  //   JSON array - [val1, val2, ...] and
  //   JSON object - {"col1":val1, "col2":val2, ...}
  // based on the top level opening bracket
  const auto first_square_bracket = std::find(first_row.begin(), first_row.end(), '[');
  const auto first_curly_bracket  = std::find(first_row.begin(), first_row.end(), '{');
  CUDF_EXPECTS(first_curly_bracket != first_row.end() || first_square_bracket != first_row.end(),
               "Input data is not a valid JSON file.");
  // If the first opening bracket is '{', assume object format
  if (first_curly_bracket < first_square_bracket) {
    // use keys as column names if input rows are objects
    auto keys_desc         = get_json_object_keys_hashes(stream);
    metadata_.column_names = keys_desc.first;
    set_column_map(std::move(keys_desc.second));
  } else {
    int cols_found = 0;
    bool quotation = false;
    for (size_t pos = 0; pos < first_row.size(); ++pos) {
      // Flip the quotation flag if current character is a quotechar
      if (first_row[pos] == opts_.quotechar) {
        quotation = !quotation;
      }
      // Check if end of a column/row
      else if (pos == first_row.size() - 1 || (!quotation && first_row[pos] == opts_.delimiter)) {
        metadata_.column_names.emplace_back(std::to_string(cols_found++));
      }
    }
  }
}

/**
 * @brief Set the data type array data member
 *
 * If user does not pass the data types, deduces types from the file content
 *
 * @param[in] stream CUDA stream used for device memory operations and kernel launches.
 */
void reader::impl::set_data_types(hipStream_t stream)
{
  auto const dtype = options_.get_dtypes();
  if (!dtype.empty()) {
    CUDF_EXPECTS(dtype.size() == metadata_.column_names.size(),
                 "Need to specify the type of each column.\n");

    // Assume that the dtype is in dictionary format only if all elements contain a colon
    const bool is_dict =
      std::all_of(std::cbegin(dtype), std::cend(dtype), [](const std::string &s) {
        return std::find(std::cbegin(s), std::cend(s), ':') != std::cend(s);
      });

    // When C++17, use std::string_view and CTAD
    auto split_on_colon = [](auto const &s) -> std::pair<std::string, std::string> {
      auto const i = s.find(":");
      auto const a = s.substr(0, i);
      auto const b = s.substr(i + 1);
      return {a, b};
    };

    if (is_dict) {
      std::map<std::string, data_type> col_type_map;
      std::transform(std::cbegin(dtype),
                     std::cend(dtype),
                     std::inserter(col_type_map, col_type_map.end()),
                     [&](auto const &ts) -> std::pair<std::string, data_type> {
                       // When C++17, use structured bindings: auto const& [col_name, type_str] = ..
                       auto split = split_on_colon(ts);
                       return {split.first, convert_string_to_dtype(split.second)};
                     });

      // Using the map here allows O(n log n) complexity
      std::transform(std::cbegin(metadata_.column_names),
                     std::cend(metadata_.column_names),
                     std::back_inserter(dtypes_),
                     [&](auto const &column_name) { return col_type_map[column_name]; });
    } else {
      std::transform(std::cbegin(dtype),
                     std::cend(dtype),
                     std::back_inserter(dtypes_),
                     [](auto const &col_dtype) { return convert_string_to_dtype(col_dtype); });
    }
  } else {
    CUDF_EXPECTS(rec_starts_.size() != 0, "No data available for data type inference.\n");
    auto const num_columns       = metadata_.column_names.size();
    auto const do_set_null_count = key_to_col_idx_map_ != nullptr;

    auto const h_column_infos = cudf::io::json::gpu::detect_data_types(
      opts_.view(),
      device_span<char const>(static_cast<char const *>(data_.data()), data_.size()),
      rec_starts_,
      do_set_null_count,
      num_columns,
      get_column_map_device_ptr(),
      stream);

    auto get_type_id = [&](auto const &cinfo) {
      auto int_count_total =
        cinfo.big_int_count + cinfo.negative_small_int_count + cinfo.positive_small_int_count;
      if (cinfo.null_count == static_cast<int>(rec_starts_.size())) {
        // Entire column is NULL; allocate the smallest amount of memory
        return type_id::INT8;
      } else if (cinfo.string_count > 0) {
        return type_id::STRING;
      } else if (cinfo.datetime_count > 0) {
        return type_id::TIMESTAMP_MILLISECONDS;
      } else if (cinfo.float_count > 0 || (int_count_total > 0 && cinfo.null_count > 0)) {
        return type_id::FLOAT64;
      } else if (cinfo.big_int_count == 0 && int_count_total != 0) {
        return type_id::INT64;
      } else if (cinfo.big_int_count != 0 && cinfo.negative_small_int_count != 0) {
        return type_id::STRING;
      } else if (cinfo.big_int_count != 0) {
        return type_id::UINT64;
      } else if (cinfo.bool_count > 0) {
        return type_id::BOOL8;
      } else {
        CUDF_FAIL("Data type detection failed.\n");
      }
    };

    std::transform(std::cbegin(h_column_infos),
                   std::cend(h_column_infos),
                   std::back_inserter(dtypes_),
                   [&](auto const &cinfo) { return data_type{get_type_id(cinfo)}; });
  }
}  // namespace json

/**
 * @brief Parse the input data and store results a table
 *
 * @param[in] stream CUDA stream used for device memory operations and kernel launches.
 *
 * @return table_with_metadata struct
 */
table_with_metadata reader::impl::convert_data_to_table(hipStream_t stream)
{
  const auto num_columns = dtypes_.size();
  const auto num_records = rec_starts_.size();

  // alloc output buffers.
  std::vector<column_buffer> out_buffers;
  for (size_t col = 0; col < num_columns; ++col) {
    out_buffers.emplace_back(dtypes_[col], num_records, true, stream, mr_);
  }

  thrust::host_vector<data_type> h_dtypes(num_columns);
  thrust::host_vector<void *> h_data(num_columns);
  thrust::host_vector<bitmask_type *> h_valid(num_columns);

  for (size_t i = 0; i < num_columns; ++i) {
    h_dtypes[i] = dtypes_[i];
    h_data[i]   = out_buffers[i].data();
    h_valid[i]  = out_buffers[i].null_mask();
  }

  rmm::device_vector<data_type> d_dtypes           = h_dtypes;
  rmm::device_vector<void *> d_data                = h_data;
  rmm::device_vector<cudf::bitmask_type *> d_valid = h_valid;
  rmm::device_vector<cudf::size_type> d_valid_counts(num_columns, 0);

  cudf::io::json::gpu::convert_json_to_columns(
    opts_.view(),
    device_span<char const>(static_cast<char const *>(data_.data()), data_.size()),
    rec_starts_,
    d_dtypes,
    get_column_map_device_ptr(),
    d_data,
    d_valid,
    d_valid_counts,
    stream);

  CUDA_TRY(hipStreamSynchronize(stream));
  CUDA_TRY(hipGetLastError());

  // postprocess columns
  auto target = make_strings_column(
    std::vector<char>{'\\', '"', '\\', '\\', '\\', 't', '\\', 'r', '\\', 'b'}, {0, 2, 4, 6, 8, 10});
  auto repl = make_strings_column({'"', '\\', '\t', '\r', '\b'}, {0, 1, 2, 3, 4, 5});

  thrust::host_vector<cudf::size_type> h_valid_counts = d_valid_counts;
  std::vector<std::unique_ptr<column>> out_columns;
  for (size_t i = 0; i < num_columns; ++i) {
    out_buffers[i].null_count() = num_records - h_valid_counts[i];

    auto out_column = make_column(out_buffers[i], stream, mr_);
    if (out_column->type().id() == type_id::STRING) {
      // Need to remove escape character in case of '\"' and '\\'
      out_columns.emplace_back(cudf::strings::detail::replace(
        out_column->view(), target->view(), repl->view(), stream, mr_));
    } else {
      out_columns.emplace_back(std::move(out_column));
    }
  }

  CUDF_EXPECTS(!out_columns.empty(), "No columns created from json input");

  return table_with_metadata{std::make_unique<table>(std::move(out_columns)), metadata_};
}

reader::impl::impl(std::unique_ptr<datasource> source,
                   std::string filepath,
                   json_reader_options const &options,
                   rmm::mr::device_memory_resource *mr)
  : source_(std::move(source)), filepath_(filepath), options_(options), mr_(mr)
{
  CUDF_EXPECTS(options_.is_enabled_lines(), "Only JSON Lines format is currently supported.\n");

  opts_.trie_true  = createSerializedTrie({"true"});
  opts_.trie_false = createSerializedTrie({"false"});
  opts_.trie_na    = createSerializedTrie({"null"});

  opts_.dayfirst = options.is_enabled_dayfirst();
}

/**
 * @brief Read an entire set or a subset of data from the source
 *
 * @param[in] range_offset Number of bytes offset from the start
 * @param[in] range_size Bytes to read; use `0` for all remaining data
 * @param[in] stream CUDA stream used for device memory operations and kernel launches.
 *
 * @return Table and its metadata
 */
table_with_metadata reader::impl::read(json_reader_options const &options, hipStream_t stream)
{
  auto range_offset = options.get_byte_range_offset();
  auto range_size   = options.get_byte_range_size();

  ingest_raw_input(range_offset, range_size);
  CUDF_EXPECTS(buffer_ != nullptr, "Ingest failed: input data is null.\n");

  decompress_input(stream);
  CUDF_EXPECTS(uncomp_data_ != nullptr, "Ingest failed: uncompressed input data is null.\n");
  CUDF_EXPECTS(uncomp_size_ != 0, "Ingest failed: uncompressed input data has zero size.\n");

  set_record_starts(stream);
  CUDF_EXPECTS(!rec_starts_.empty(), "Error enumerating records.\n");

  upload_data_to_device(stream);
  CUDF_EXPECTS(data_.size() != 0, "Error uploading input data to the GPU.\n");

  set_column_names(stream);
  CUDF_EXPECTS(!metadata_.column_names.empty(), "Error determining column names.\n");

  set_data_types(stream);
  CUDF_EXPECTS(!dtypes_.empty(), "Error in data type detection.\n");

  return convert_data_to_table(stream);
}

// Forward to implementation
reader::reader(std::vector<std::string> const &filepaths,
               json_reader_options const &options,
               rmm::mr::device_memory_resource *mr)
{
  CUDF_EXPECTS(filepaths.size() == 1, "Only a single source is currently supported.");
  // Delay actual instantiation of data source until read to allow for
  // partial memory mapping of file using byte ranges
  _impl = std::make_unique<impl>(nullptr, filepaths[0], options, mr);
}

// Forward to implementation
reader::reader(std::vector<std::unique_ptr<cudf::io::datasource>> &&sources,
               json_reader_options const &options,
               rmm::mr::device_memory_resource *mr)
{
  CUDF_EXPECTS(sources.size() == 1, "Only a single source is currently supported.");
  _impl = std::make_unique<impl>(std::move(sources[0]), "", options, mr);
}

// Destructor within this translation unit
reader::~reader() = default;

// Forward to implementation
table_with_metadata reader::read(json_reader_options const &options, hipStream_t stream)
{
  return table_with_metadata{_impl->read(options, stream)};
}
}  // namespace json
}  // namespace detail
}  // namespace io
}  // namespace cudf
