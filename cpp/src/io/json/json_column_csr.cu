#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2022-2024, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "json_utils.hpp"
#include "nested_json.hpp"

#include <cudf/column/column_factories.hpp>
#include <cudf/detail/null_mask.hpp>
#include <cudf/detail/nvtx/ranges.hpp>
#include <cudf/detail/utilities/vector_factories.hpp>
#include <cudf/detail/utilities/visitor_overload.hpp>
#include <cudf/strings/strings_column_view.hpp>
#include <cudf/types.hpp>
#include <cudf/utilities/error.hpp>
#include <cudf/utilities/span.hpp>

#include <rmm/cuda_stream_view.hpp>
#include <rmm/device_uvector.hpp>
#include <rmm/exec_policy.hpp>
#include <rmm/resource_ref.hpp>

#include <cuda/atomic>
#include <cuda/functional>
#include <thrust/adjacent_difference.h>
#include <thrust/binary_search.h>
#include <thrust/count.h>
#include <thrust/for_each.h>
#include <thrust/functional.h>
#include <thrust/gather.h>
#include <thrust/iterator/constant_iterator.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/iterator/discard_iterator.h>
#include <thrust/iterator/permutation_iterator.h>
#include <thrust/iterator/zip_iterator.h>
#include <thrust/logical.h>
#include <thrust/partition.h>
#include <thrust/reduce.h>
#include <thrust/scan.h>
#include <thrust/sort.h>
#include <thrust/transform.h>
#include <thrust/unique.h>

namespace cudf::io::json::experimental::detail {

struct device_json_column_properties_size {
  rmm::device_uvector<NodeIndexT> outcol_nodes;
  size_t string_offsets_size = 0;
  size_t string_lengths_size = 0;
  size_t child_offsets_size  = 0;
  size_t num_rows_size       = 0;
};

device_json_column_properties_size estimate_device_json_column_size(
  rmm::device_uvector<NodeIndexT> const& rowidx,
  rmm::device_uvector<NodeIndexT> const& colidx,
  rmm::device_uvector<NodeT> const& categories,
  cudf::io::json_reader_options reader_options,
  rmm::cuda_stream_view stream)
{
  // What are the cases in which estimation works?
  CUDF_EXPECTS(reader_options.is_enabled_mixed_types_as_string() == false,
               "mixed type as string has not yet been implemented");
  CUDF_EXPECTS(reader_options.is_enabled_prune_columns() == false,
               "column pruning has not yet been implemented");
  // traverse the column tree
  auto num_columns = rowidx.size() - 1;

  // 1. TODO: removing NC_ERR nodes and their descendants i.e.
  // removing the entire subtree rooted at the nodes with category NC_ERR
  // for now, we just assert that there are indeed no error nodes
  auto num_err_nodes = thrust::count_if(
    rmm::exec_policy(stream), categories.begin(), categories.end(), [] __device__(auto const ctg) {
      return ctg == NC_ERR;
    });
  CUDF_EXPECTS(num_err_nodes == 0, "oops, there are some error nodes in the column tree!");

  // 2. Let's do some validation of the column tree based on its properties.
  // We will be using these properties to filter nodes later on.
  // ===========================================================================
  // (i) Every node v is of type string, val, field name, list or struct.
  // (ii) String and val cannot have any children i.e. they can only be leaf nodes
  // (iii) If v is a field name, it can have struct, list, string and val as children.
  // (iv) If v is a struct, it can have a field name as child
  // (v) If v is a list, it can have string, val, list or struct as child
  // (vi) There can only be at most one string and one val child for a given node, but many struct,
  // list and field name children. (vii) When mixed type support is disabled -
  //       (a) A mix of lists and structs in the same column is not supported i.e a field name and
  //       list node cannot have both list and struct as children (b) If there is a mix of str/val
  //       and list/struct in the same column, then str/val is discarded

  // Validation of (vii)(a)
  auto num_field_and_list_nodes = thrust::count_if(
    rmm::exec_policy(stream), categories.begin(), categories.end(), [] __device__(auto const ctg) {
      return ctg == NC_FN || ctg == NC_LIST;
    });
  rmm::device_uvector<NodeIndexT> field_and_list_nodes(num_field_and_list_nodes, stream);
  thrust::partition_copy(rmm::exec_policy(stream),
                         thrust::make_counting_iterator(0),
                         thrust::make_counting_iterator(0) + num_columns,
                         field_and_list_nodes.begin(),
                         thrust::make_discard_iterator(),
                         [categories = categories.begin()] __device__(NodeIndexT node) {
                           return categories[node] == NC_LIST || categories[node] == NC_FN;
                         });
  bool is_valid_tree = thrust::all_of(
    rmm::exec_policy(stream),
    field_and_list_nodes.begin(),
    field_and_list_nodes.end(),
    [rowidx = rowidx.begin(), colidx = colidx.begin(), categories = categories.begin()] __device__(
      NodeIndexT node) {
      NodeIndexT first_child_pos = rowidx[node] + 1;
      NodeIndexT last_child_pos  = rowidx[node + 1] - 1;
      bool has_struct_child      = false;
      bool has_list_child        = false;
      for (NodeIndexT child_pos = first_child_pos; child_pos <= last_child_pos; child_pos++) {
        if (categories[colidx[child_pos]] == NC_STRUCT) has_struct_child = true;
        if (categories[colidx[child_pos]] == NC_LIST) has_list_child = true;
      }
      return !has_struct_child && !has_list_child;
    });

  CUDF_EXPECTS(is_valid_tree,
               "Invalidating property 7a i.e. mix of LIST and STRUCT in same column is not "
               "supported when mixed type support is disabled");

  // Validation of (vii)(b) i.e. ignore_vals in previous implementation
  // We need to identify leaf nodes that have non-leaf sibling nodes
  // i.e. we need to ignore leaf nodes at level above the last level
  // idea: leaf nodes have adjacency 1. So if there is an adjacency 1 inbetween non-one
  // adjacencies, then found the leaf node. Corner case: consider the last set of consecutive
  // ones. If the leftmost of those ones (say node u) has a non-leaf sibling
  // (can be found by looking at the adjacencies of the siblings
  // (which are in turn found from the colidx of the parent u), then this leaf node should be
  // ignored, otherwise all good.
  rmm::device_uvector<NodeIndexT> adjacency(
    num_columns + 1,
    stream);  // since adjacent_difference requires that the output have the same length as input
  thrust::adjacent_difference(
    rmm::exec_policy(stream), rowidx.begin(), rowidx.end(), adjacency.begin());
  auto num_leaf_nodes = thrust::count_if(rmm::exec_policy(stream),
                                         adjacency.begin() + 1,
                                         adjacency.end(),
                                         [] __device__(auto const adj) { return adj == 1; });
  rmm::device_uvector<NodeIndexT> leaf_nodes(num_leaf_nodes, stream);
  thrust::copy_if(
    rmm::exec_policy(stream),
    thrust::make_counting_iterator(0),
    thrust::make_counting_iterator(0) + num_columns,
    leaf_nodes.begin(),
    [adjacency = adjacency.begin()] __device__(size_t node) { return adjacency[node] == 1; });

  auto rev_node_it = thrust::make_reverse_iterator(thrust::make_counting_iterator(0) + num_columns);
  auto rev_leaf_nodes_it = thrust::make_reverse_iterator(leaf_nodes.begin());
  auto is_leftmost_leaf  = thrust::mismatch(
    rmm::exec_policy(stream), rev_node_it, rev_node_it + num_columns, rev_leaf_nodes_it);
  // the node number that could be the leftmost leaf node is given by u = *(is_leftmost_leaf.second
  // - 1)
  NodeIndexT leftmost_leaf_node = leaf_nodes.element(
    num_leaf_nodes - thrust::distance(rev_leaf_nodes_it, is_leftmost_leaf.second - 1) - 1, stream);

  // upper_bound search for u in rowidx for parent node v. Now check if any of the other child nodes
  // of v is non-leaf i.e check if u is the first child of v. If yes, then leafmost_leaf_node is
  // the leftmost leaf node. Otherwise, discard all children of v after and including u

  auto parent_it =
    thrust::upper_bound(rmm::exec_policy(stream), rowidx.begin(), rowidx.end(), leftmost_leaf_node);
  NodeIndexT parent           = thrust::distance(rowidx.begin(), parent_it - 1);
  NodeIndexT parent_adj_start = rowidx.element(parent, stream);
  NodeIndexT parent_adj_end   = rowidx.element(parent + 1, stream);
  auto childnum_it            = thrust::lower_bound(rmm::exec_policy(stream),
                                         colidx.begin() + parent_adj_start,
                                         colidx.begin() + parent_adj_end,
                                         leftmost_leaf_node);

  auto retained_leaf_nodes_it = leaf_nodes.begin() + num_leaf_nodes -
                                thrust::distance(rev_leaf_nodes_it, is_leftmost_leaf.second - 1) -
                                1;
  if (childnum_it != colidx.begin() + parent_adj_start + 1) {
    // discarding from u to last child of parent
    retained_leaf_nodes_it += thrust::distance(childnum_it, colidx.begin() + parent_adj_end);
  }
  // now, all nodes from leaf_nodes.begin() to retained_leaf_nodes_it need to be discarded i.e. they
  // are part of ignore_vals

  // (Optional?) TODO: Validation of the remaining column tree properties

  rmm::device_uvector<NodeIndexT> outcol_nodes(num_columns, stream);
  return device_json_column_properties_size{std::move(outcol_nodes)};
}

/**
 * @brief Reduces node tree representation to column tree CSR representation.
 *
 * @param tree Node tree representation of JSON string
 * @param original_col_ids Column ids of nodes
 * @param sorted_col_ids Sorted column ids of nodes
 * @param ordered_node_ids Node ids of nodes sorted by column ids
 * @param row_offsets Row offsets of nodes
 * @param is_array_of_arrays Whether the tree is an array of arrays
 * @param row_array_parent_col_id Column id of row array, if is_array_of_arrays is true
 * @param stream CUDA stream used for device memory operations and kernel launches
 * @return A tuple of column tree representation of JSON string, column ids of columns, and
 * max row offsets of columns
 */
std::tuple<column_tree_csr, rmm::device_uvector<size_type>> reduce_to_column_tree_csr(
  tree_meta_t& tree,
  device_span<NodeIndexT> original_col_ids,
  device_span<NodeIndexT> sorted_col_ids,
  device_span<NodeIndexT> ordered_node_ids,
  device_span<size_type> row_offsets,
  bool is_array_of_arrays,
  NodeIndexT const row_array_parent_col_id,
  cudf::io::json_reader_options const& reader_options,
  rmm::cuda_stream_view stream)
{
  CUDF_FUNC_RANGE();
  // 1. column count for allocation
  auto const num_columns =
    thrust::unique_count(rmm::exec_policy(stream), sorted_col_ids.begin(), sorted_col_ids.end());

  rmm::device_uvector<size_type> unique_node_ids(num_columns, stream);
  rmm::device_uvector<size_type> csr_unique_node_ids(num_columns, stream);
  rmm::device_uvector<TreeDepthT> column_levels(num_columns, stream);
  thrust::unique_by_key_copy(rmm::exec_policy(stream),
                             sorted_col_ids.begin(),
                             sorted_col_ids.end(),
                             ordered_node_ids.begin(),
                             thrust::make_discard_iterator(),
                             unique_node_ids.begin());
  thrust::copy_n(
    rmm::exec_policy(stream),
    thrust::make_permutation_iterator(tree.node_levels.begin(), unique_node_ids.begin()),
    unique_node_ids.size(),
    column_levels.begin());
  auto [sorted_column_levels, sorted_column_levels_order] =
    cudf::io::json::detail::stable_sorted_key_order<size_t, TreeDepthT>(column_levels, stream);

  // 2. reduce_by_key {col_id}, {row_offset}, max.
  rmm::device_uvector<NodeIndexT> unique_col_ids(num_columns, stream);
  rmm::device_uvector<size_type> max_row_offsets(num_columns, stream);
  rmm::device_uvector<NodeIndexT> csr_unique_col_ids(num_columns, stream);
  rmm::device_uvector<size_type> csr_max_row_offsets(num_columns, stream);
  auto ordered_row_offsets =
    thrust::make_permutation_iterator(row_offsets.begin(), ordered_node_ids.begin());
  thrust::reduce_by_key(rmm::exec_policy(stream),
                        sorted_col_ids.begin(),
                        sorted_col_ids.end(),
                        ordered_row_offsets,
                        unique_col_ids.begin(),
                        max_row_offsets.begin(),
                        thrust::equal_to<size_type>(),
                        thrust::maximum<size_type>());

  // 3. reduce_by_key {col_id}, {node_categories} - custom opp (*+v=*, v+v=v, *+#=E)
  rmm::device_uvector<NodeT> column_categories(num_columns, stream);
  rmm::device_uvector<NodeT> csr_column_categories(num_columns, stream);
  thrust::reduce_by_key(
    rmm::exec_policy(stream),
    sorted_col_ids.begin(),
    sorted_col_ids.end(),
    thrust::make_permutation_iterator(tree.node_categories.begin(), ordered_node_ids.begin()),
    unique_col_ids.begin(),
    column_categories.begin(),
    thrust::equal_to<size_type>(),
    [] __device__(NodeT type_a, NodeT type_b) -> NodeT {
      auto is_a_leaf = (type_a == NC_VAL || type_a == NC_STR);
      auto is_b_leaf = (type_b == NC_VAL || type_b == NC_STR);
      // (v+v=v, *+*=*,  *+v=*, *+#=E, NESTED+VAL=NESTED)
      // *+*=*, v+v=v
      if (type_a == type_b) {
        return type_a;
      } else if (is_a_leaf) {
        // *+v=*, N+V=N
        // STRUCT/LIST + STR/VAL = STRUCT/LIST, STR/VAL + FN = ERR, STR/VAL + STR = STR
        return type_b == NC_FN ? NC_ERR : (is_b_leaf ? NC_STR : type_b);
      } else if (is_b_leaf) {
        return type_a == NC_FN ? NC_ERR : (is_a_leaf ? NC_STR : type_a);
      }
      // *+#=E
      return NC_ERR;
    });

  auto csr_permutation_it = thrust::make_zip_iterator(
    thrust::make_permutation_iterator(unique_node_ids.begin(), sorted_column_levels_order.begin()),
    thrust::make_permutation_iterator(unique_col_ids.begin(), sorted_column_levels_order.begin()),
    thrust::make_permutation_iterator(max_row_offsets.begin(), sorted_column_levels_order.begin()),
    thrust::make_permutation_iterator(column_categories.begin(),
                                      sorted_column_levels_order.begin()));
  thrust::copy(rmm::exec_policy(stream),
               csr_permutation_it,
               csr_permutation_it + num_columns,
               thrust::make_zip_iterator(csr_unique_node_ids.begin(),
                                         csr_unique_col_ids.begin(),
                                         csr_max_row_offsets.begin(),
                                         csr_column_categories.begin()));

  // 4. unique_copy parent_node_ids, ranges
  rmm::device_uvector<NodeIndexT> csr_parent_col_ids(num_columns, stream);
  rmm::device_uvector<SymbolOffsetT> csr_col_range_begin(num_columns, stream);  // Field names
  rmm::device_uvector<SymbolOffsetT> csr_col_range_end(num_columns, stream);
  thrust::copy_n(
    rmm::exec_policy(stream),
    thrust::make_zip_iterator(
      thrust::make_permutation_iterator(tree.parent_node_ids.begin(), csr_unique_node_ids.begin()),
      thrust::make_permutation_iterator(tree.node_range_begin.begin(), csr_unique_node_ids.begin()),
      thrust::make_permutation_iterator(tree.node_range_end.begin(), csr_unique_node_ids.begin())),
    csr_unique_node_ids.size(),
    thrust::make_zip_iterator(
      csr_parent_col_ids.begin(), csr_col_range_begin.begin(), csr_col_range_end.begin()));

  // convert parent_node_ids to parent_col_ids
  thrust::transform(
    rmm::exec_policy(stream),
    csr_parent_col_ids.begin(),
    csr_parent_col_ids.end(),
    csr_parent_col_ids.begin(),
    [col_ids = original_col_ids.begin()] __device__(auto parent_node_id) -> size_type {
      return parent_node_id == parent_node_sentinel ? parent_node_sentinel
                                                    : col_ids[parent_node_id];
    });

  /*
    CSR construction:
    1. Sort column levels and get their ordering
    2. For each column node coln iterated according to sorted_column_levels; do
        a. Find nodes that have coln as the parent node -> set adj_coln
        b. row idx[coln] = size of adj_coln + 1
        c. col idx[coln] = adj_coln U {parent_col_id[coln]}
  */

  rmm::device_uvector<NodeIndexT> rowidx(num_columns + 1, stream);
  thrust::fill(rmm::exec_policy(stream), rowidx.begin(), rowidx.end(), 0);

  // Note that the first element of csr_parent_col_ids is -1 (parent_node_sentinel)
  // children adjacency
  auto num_non_leaf_columns = thrust::unique_count(
    rmm::exec_policy(stream), csr_parent_col_ids.begin() + 1, csr_parent_col_ids.end());
  thrust::reduce_by_key(rmm::exec_policy(stream),
                        csr_parent_col_ids.begin() + 1,
                        csr_parent_col_ids.end(),
                        thrust::make_constant_iterator(1),
                        thrust::make_discard_iterator(),
                        rowidx.begin() + 1,
                        thrust::equal_to<TreeDepthT>());
  thrust::inclusive_scan(
    rmm::exec_policy(stream), rowidx.begin() + 1, rowidx.end(), rowidx.begin() + 1);
  // overwrite the csr_parent_col_ids with the col ids in the csr tree
  thrust::fill(rmm::exec_policy(stream), csr_parent_col_ids.begin(), csr_parent_col_ids.end(), -1);
  thrust::scatter(rmm::exec_policy(stream),
                  thrust::make_counting_iterator(0),
                  thrust::make_counting_iterator(0) + num_non_leaf_columns,
                  rowidx.begin(),
                  csr_parent_col_ids.begin() + 1);
  thrust::inclusive_scan(rmm::exec_policy(stream),
                         csr_parent_col_ids.begin(),
                         csr_parent_col_ids.end(),
                         csr_parent_col_ids.begin(),
                         thrust::maximum<NodeIndexT>{});
  // We are discarding the parent of the root node. Add the parent adjacency. Since we have already
  // performed the scan, we use a counting iterator to add
  thrust::transform(rmm::exec_policy(stream),
                    rowidx.begin() + 2,
                    rowidx.end(),
                    thrust::make_counting_iterator(1),
                    rowidx.begin() + 2,
                    thrust::plus<NodeIndexT>());

  rmm::device_uvector<NodeIndexT> colidx((num_columns - 1) * 2, stream);
  thrust::fill(rmm::exec_policy(stream), colidx.begin(), colidx.end(), 0);
  // Skip the parent of root node
  thrust::scatter(rmm::exec_policy(stream),
                  csr_parent_col_ids.begin() + 1,
                  csr_parent_col_ids.end(),
                  rowidx.begin() + 1,
                  colidx.begin());
  // excluding root node
  rmm::device_uvector<NodeIndexT> map(num_columns - 1, stream);
  thrust::fill(rmm::exec_policy(stream), map.begin(), map.end(), 1);
  thrust::inclusive_scan_by_key(rmm::exec_policy(stream),
                                csr_parent_col_ids.begin() + 1,
                                csr_parent_col_ids.end(),
                                map.begin(),
                                map.begin());
  thrust::for_each(rmm::exec_policy(stream),
                   thrust::make_counting_iterator(1),
                   thrust::make_counting_iterator(1) + num_columns - 1,
                   [rowidx             = rowidx.begin(),
                    map                = map.begin(),
                    csr_parent_col_ids = csr_parent_col_ids.begin()] __device__(auto i) {
                     auto csr_parent_col_id = csr_parent_col_ids[i];
                     if (csr_parent_col_id == 0)
                       map[i - 1]--;
                     else
                       map[i - 1] += rowidx[csr_parent_col_id];
                   });
  thrust::scatter(rmm::exec_policy(stream),
                  thrust::make_counting_iterator(1),
                  thrust::make_counting_iterator(1) + num_columns - 1,
                  map.begin(),
                  colidx.begin());

  // condition is true if parent is not a list, or sentinel/root
  // Special case to return true if parent is a list and is_array_of_arrays is true
  auto is_non_list_parent = [column_categories = column_categories.begin(),
                             is_array_of_arrays,
                             row_array_parent_col_id] __device__(auto parent_col_id) -> bool {
    return !(parent_col_id == parent_node_sentinel ||
             column_categories[parent_col_id] == NC_LIST &&
               (!is_array_of_arrays || parent_col_id != row_array_parent_col_id));
  };
  // Mixed types in List children go to different columns,
  // so all immediate children of list column should have same max_row_offsets.
  //   create list's children max_row_offsets array. (initialize to zero)
  //   atomicMax on  children max_row_offsets array.
  //   gather the max_row_offsets from children row offset array.
  {
    rmm::device_uvector<NodeIndexT> list_parents_children_max_row_offsets(num_columns, stream);
    thrust::fill(rmm::exec_policy(stream),
                 list_parents_children_max_row_offsets.begin(),
                 list_parents_children_max_row_offsets.end(),
                 0);
    thrust::for_each(rmm::exec_policy(stream),
                     csr_unique_col_ids.begin(),
                     csr_unique_col_ids.end(),
                     [csr_column_categories = csr_column_categories.begin(),
                      csr_parent_col_ids    = csr_parent_col_ids.begin(),
                      csr_max_row_offsets   = csr_max_row_offsets.begin(),
                      list_parents_children_max_row_offsets =
                        list_parents_children_max_row_offsets.begin()] __device__(auto col_id) {
                       auto csr_parent_col_id = csr_parent_col_ids[col_id];
                       if (csr_parent_col_id != parent_node_sentinel and
                           csr_column_categories[csr_parent_col_id] == node_t::NC_LIST) {
                         cuda::atomic_ref<NodeIndexT, cuda::thread_scope_device> ref{
                           *(list_parents_children_max_row_offsets + csr_parent_col_id)};
                         ref.fetch_max(csr_max_row_offsets[col_id],
                                       cuda::std::memory_order_relaxed);
                       }
                     });
    thrust::gather_if(
      rmm::exec_policy(stream),
      csr_parent_col_ids.begin(),
      csr_parent_col_ids.end(),
      csr_parent_col_ids.begin(),
      list_parents_children_max_row_offsets.begin(),
      csr_max_row_offsets.begin(),
      [csr_column_categories = csr_column_categories.begin()] __device__(size_type parent_col_id) {
        return parent_col_id != parent_node_sentinel and
               csr_column_categories[parent_col_id] == node_t::NC_LIST;
      });
  }

  // copy lists' max_row_offsets to children.
  // all structs should have same size.
  thrust::transform_if(
    rmm::exec_policy(stream),
    csr_unique_col_ids.begin(),
    csr_unique_col_ids.end(),
    csr_max_row_offsets.begin(),
    [csr_column_categories = csr_column_categories.begin(),
     is_non_list_parent,
     csr_parent_col_ids  = csr_parent_col_ids.begin(),
     csr_max_row_offsets = csr_max_row_offsets.begin()] __device__(size_type col_id) {
      auto parent_col_id = csr_parent_col_ids[col_id];
      // condition is true if parent is not a list, or sentinel/root
      while (is_non_list_parent(parent_col_id)) {
        col_id        = parent_col_id;
        parent_col_id = csr_parent_col_ids[parent_col_id];
      }
      return csr_max_row_offsets[col_id];
    },
    [csr_column_categories = csr_column_categories.begin(),
     is_non_list_parent,
     parent_col_ids = csr_parent_col_ids.begin()] __device__(size_type col_id) {
      auto parent_col_id = parent_col_ids[col_id];
      // condition is true if parent is not a list, or sentinel/root
      return is_non_list_parent(parent_col_id);
    });

  // For Struct and List (to avoid copying entire strings when mixed type as string is enabled)
  thrust::transform_if(
    rmm::exec_policy(stream),
    csr_col_range_begin.begin(),
    csr_col_range_begin.end(),
    csr_column_categories.begin(),
    csr_col_range_end.begin(),
    [] __device__(auto i) { return i + 1; },
    [] __device__(NodeT type) { return type == NC_STRUCT || type == NC_LIST; });

  auto size_estimates =
    estimate_device_json_column_size(rowidx, colidx, csr_column_categories, reader_options, stream);

  return std::tuple{column_tree_csr{std::move(rowidx),
                                    std::move(colidx),
                                    std::move(csr_unique_col_ids),
                                    std::move(csr_column_categories),
                                    std::move(csr_col_range_begin),
                                    std::move(csr_col_range_end)},
                    std::move(csr_max_row_offsets)};
}

/**
 * @brief Constructs `d_json_column` from node tree representation
 * Newly constructed columns are insert into `root`'s children.
 * `root` must be a list type.
 *
 * @param input Input JSON string device data
 * @param tree Node tree representation of the JSON string
 * @param col_ids Column ids of the nodes in the tree
 * @param row_offsets Row offsets of the nodes in the tree
 * @param root Root node of the `d_json_column` tree
 * @param is_array_of_arrays Whether the tree is an array of arrays
 * @param options Parsing options specifying the parsing behaviour
 * options affecting behaviour are
 *   is_enabled_lines: Whether the input is a line-delimited JSON
 *   is_enabled_mixed_types_as_string: Whether to enable reading mixed types as string
 * @param stream CUDA stream used for device memory operations and kernel launches
 * @param mr Device memory resource used to allocate the device memory
 * of child_offets and validity members of `d_json_column`
 */
void make_device_json_column_csr(device_span<SymbolT const> input,
                                 tree_meta_t& tree,
                                 device_span<NodeIndexT> col_ids,
                                 device_span<size_type> row_offsets,
                                 device_json_column& root,
                                 bool is_array_of_arrays,
                                 cudf::io::json_reader_options const& options,
                                 rmm::cuda_stream_view stream,
                                 rmm::device_async_resource_ref mr)
{
  CUDF_FUNC_RANGE();
  bool const is_enabled_lines = options.is_enabled_lines();
  auto const num_nodes        = col_ids.size();
  rmm::device_uvector<NodeIndexT> sorted_col_ids(col_ids.size(), stream);  // make a copy
  thrust::copy(rmm::exec_policy(stream), col_ids.begin(), col_ids.end(), sorted_col_ids.begin());

  // sort by {col_id} on {node_ids} stable
  rmm::device_uvector<NodeIndexT> node_ids(col_ids.size(), stream);
  thrust::sequence(rmm::exec_policy(stream), node_ids.begin(), node_ids.end());
  thrust::stable_sort_by_key(
    rmm::exec_policy(stream), sorted_col_ids.begin(), sorted_col_ids.end(), node_ids.begin());

  NodeIndexT const row_array_parent_col_id = [&]() {
    NodeIndexT value = parent_node_sentinel;
    if (!col_ids.empty()) {
      auto const list_node_index = is_enabled_lines ? 0 : 1;
      CUDF_CUDA_TRY(hipMemcpyAsync(&value,
                                    col_ids.data() + list_node_index,
                                    sizeof(NodeIndexT),
                                    hipMemcpyDefault,
                                    stream.value()));
      stream.synchronize();
    }
    return value;
  }();

  // 1. gather column information.
  auto [d_column_tree, d_max_row_offsets] = reduce_to_column_tree_csr(tree,
                                                                      col_ids,
                                                                      sorted_col_ids,
                                                                      node_ids,
                                                                      row_offsets,
                                                                      is_array_of_arrays,
                                                                      row_array_parent_col_id,
                                                                      options,
                                                                      stream);

  CUDF_EXPECTS(is_array_of_arrays == false, "array of arrays has not yet been implemented");
  CUDF_EXPECTS(options.is_enabled_mixed_types_as_string() == false,
               "mixed type as string has not yet been implemented");
  CUDF_EXPECTS(options.is_enabled_prune_columns() == false,
               "column pruning has not yet been implemented");
}

}  // namespace cudf::io::json::experimental::detail
