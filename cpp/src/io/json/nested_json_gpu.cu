#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2022-2024, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "io/fst/logical_stack.cuh"
#include "io/fst/lookup_tables.cuh"
#include "io/utilities/parsing_utils.cuh"
#include "io/utilities/string_parsing.hpp"
#include "nested_json.hpp"

#include <cudf/column/column_factories.hpp>
#include <cudf/detail/nvtx/ranges.hpp>
#include <cudf/detail/utilities/vector_factories.hpp>
#include <cudf/detail/utilities/visitor_overload.hpp>
#include <cudf/detail/valid_if.cuh>
#include <cudf/io/detail/tokenize_json.hpp>
#include <cudf/io/json.hpp>
#include <cudf/table/table.hpp>
#include <cudf/types.hpp>
#include <cudf/utilities/bit.hpp>
#include <cudf/utilities/error.hpp>
#include <cudf/utilities/span.hpp>

#include <rmm/device_scalar.hpp>
#include <rmm/device_uvector.hpp>
#include <rmm/exec_policy.hpp>
#include <rmm/resource_ref.hpp>

#include <thrust/device_vector.h>
#include <thrust/iterator/discard_iterator.h>
#include <thrust/iterator/transform_iterator.h>
#include <thrust/iterator/zip_iterator.h>
#include <thrust/transform.h>
#include <thrust/tuple.h>

#include <limits>
#include <stack>

// Debug print flag
#ifndef NJP_DEBUG_PRINT
// #define NJP_DEBUG_PRINT
#endif

namespace {

/**
 * @brief While parsing the token stream, we use a stack of tree_nodes to maintain all the
 * information about the data path that is relevant.
 */
struct tree_node {
  // The column that this node is associated with
  cudf::io::json::json_column* column;

  // The row offset that this node belongs to within the given column
  uint32_t row_index;

  // Selected child column
  // E.g., if this is a struct node, and we subsequently encountered the field name "a", then this
  // point's to the struct's "a" child column
  cudf::io::json::json_column* current_selected_col = nullptr;

  std::size_t num_children = 0;
};

/**
 * @brief Verifies that the JSON input can be handled without corrupted data due to offset
 * overflows.
 *
 * @param input_size The JSON inputs size in bytes
 */
void check_input_size(std::size_t input_size)
{
  // Transduce() writes symbol offsets that may be as large input_size-1
  CUDF_EXPECTS(input_size == 0 ||
                 (input_size - 1) <= std::numeric_limits<cudf::io::json::SymbolOffsetT>::max(),
               "Given JSON input is too large");
}
}  // namespace

namespace cudf::io::json {

// FST to help fixing the stack context of characters that follow the first record on each JSON line
namespace fix_stack_of_excess_chars {

// Type used to represent the target state in the transition table
using StateT = char;

// Type used to represent a symbol group id
using SymbolGroupT = uint8_t;

/**
 * @brief Definition of the DFA's states.
 */
enum class dfa_states : StateT {
  // Before the first record on the JSON line
  BEFORE,
  // Within the first record on the JSON line
  WITHIN,
  // Excess data that follows the first record on the JSON line
  EXCESS,
  // Total number of states
  NUM_STATES
};

/**
 * @brief Definition of the symbol groups
 */
enum class dfa_symbol_group_id : SymbolGroupT {
  ROOT,              ///< Symbol for root stack context
  DELIMITER,         ///< Line delimiter symbol group
  OTHER,             ///< Symbol group that implicitly matches all other tokens
  NUM_SYMBOL_GROUPS  ///< Total number of symbol groups
};

constexpr auto TT_NUM_STATES     = static_cast<StateT>(dfa_states::NUM_STATES);
constexpr auto NUM_SYMBOL_GROUPS = static_cast<uint32_t>(dfa_symbol_group_id::NUM_SYMBOL_GROUPS);

/**
 * @brief Function object to map (input_symbol,stack_context) tuples to a symbol group.
 */
struct SymbolPairToSymbolGroupId {
  SymbolT delimiter = '\n';
  CUDF_HOST_DEVICE SymbolGroupT operator()(thrust::tuple<SymbolT, StackSymbolT> symbol) const
  {
    auto const input_symbol = thrust::get<0>(symbol);
    auto const stack_symbol = thrust::get<1>(symbol);
    return static_cast<SymbolGroupT>(
      input_symbol == delimiter
        ? dfa_symbol_group_id::DELIMITER
        : (stack_symbol == '_' ? dfa_symbol_group_id::ROOT : dfa_symbol_group_id::OTHER));
  }
};

/**
 * @brief Translation function object that fixes the stack context of excess data that follows after
 * the first JSON record on each line.
 */
struct TransduceInputOp {
  template <typename RelativeOffsetT, typename SymbolT>
  constexpr CUDF_HOST_DEVICE StackSymbolT operator()(StateT const state_id,
                                                     SymbolGroupT const match_id,
                                                     RelativeOffsetT const relative_offset,
                                                     SymbolT const read_symbol) const
  {
    if (state_id == static_cast<StateT>(dfa_states::EXCESS)) { return '_'; }
    return thrust::get<1>(read_symbol);
  }

  template <typename SymbolT>
  constexpr CUDF_HOST_DEVICE int32_t operator()(StateT const state_id,
                                                SymbolGroupT const match_id,
                                                SymbolT const read_symbol) const
  {
    constexpr int32_t single_output_item = 1;
    return single_output_item;
  }
};

// Aliases for readability of the transition table
constexpr auto TT_BEFORE = dfa_states::BEFORE;
constexpr auto TT_INSIDE = dfa_states::WITHIN;
constexpr auto TT_EXCESS = dfa_states::EXCESS;

// Transition table
std::array<std::array<dfa_states, NUM_SYMBOL_GROUPS>, TT_NUM_STATES> constexpr transition_table{
  {/* IN_STATE            ROOT      NEWLINE     OTHER */
   /* TT_BEFORE    */ {{TT_BEFORE, TT_BEFORE, TT_INSIDE}},
   /* TT_INSIDE    */ {{TT_EXCESS, TT_BEFORE, TT_INSIDE}},
   /* TT_EXCESS    */ {{TT_EXCESS, TT_BEFORE, TT_EXCESS}}}};

// The DFA's starting state
constexpr auto start_state = static_cast<StateT>(dfa_states::BEFORE);
}  // namespace fix_stack_of_excess_chars

// FST to prune tokens of invalid lines for recovering JSON lines format
namespace token_filter {

// Type used to represent the target state in the transition table
using StateT = char;

// Type used to represent a symbol group id
using SymbolGroupT = uint8_t;

/**
 * @brief Definition of the DFA's states
 */
enum class dfa_states : StateT { VALID, INVALID, NUM_STATES };

// Aliases for readability of the transition table
constexpr auto TT_INV = dfa_states::INVALID;
constexpr auto TT_VLD = dfa_states::VALID;

/**
 * @brief Definition of the symbol groups
 */
enum class dfa_symbol_group_id : SymbolGroupT {
  ERROR,             ///< Error token symbol group
  DELIMITER,         ///< Record / line delimiter symbol group
  OTHER_SYMBOLS,     ///< Symbol group that implicitly matches all other tokens
  NUM_SYMBOL_GROUPS  ///< Total number of symbol groups
};

constexpr auto TT_NUM_STATES     = static_cast<StateT>(dfa_states::NUM_STATES);
constexpr auto NUM_SYMBOL_GROUPS = static_cast<uint32_t>(dfa_symbol_group_id::NUM_SYMBOL_GROUPS);

// Lookup table to map an input symbol (i.e., a token) to a symbol group
std::array<std::vector<PdaTokenT>, NUM_SYMBOL_GROUPS - 1> const symbol_groups{{
  {static_cast<PdaTokenT>(token_t::ErrorBegin)},  // Symbols mapping to ERROR
  {static_cast<PdaTokenT>(token_t::LineEnd)}      // Symbols mapping to DELIMITER
}};

/**
 * @brief Function object to map (token,token_index) tuples to a symbol group.
 */
struct UnwrapTokenFromSymbolOp {
  template <typename SymbolGroupLookupTableT>
  CUDF_HOST_DEVICE SymbolGroupT operator()(SymbolGroupLookupTableT const& sgid_lut,
                                           thrust::tuple<PdaTokenT, SymbolOffsetT> symbol) const
  {
    PdaTokenT const token_type = thrust::get<0>(symbol);
    return sgid_lut.lookup(token_type);
  }
};

/**
 * @brief Translation function object that discards line delimiter tokens and tokens belonging to
 * invalid lines.
 */
struct TransduceToken {
  template <typename RelativeOffsetT, typename SymbolT>
  constexpr CUDF_HOST_DEVICE SymbolT operator()(StateT const state_id,
                                                SymbolGroupT const match_id,
                                                RelativeOffsetT const relative_offset,
                                                SymbolT const read_symbol) const
  {
    const bool is_end_of_invalid_line =
      (state_id == static_cast<StateT>(TT_INV) &&
       match_id == static_cast<SymbolGroupT>(dfa_symbol_group_id::DELIMITER));

    if (is_end_of_invalid_line) {
      return relative_offset == 0 ? SymbolT{token_t::StructEnd, 0}
                                  : SymbolT{token_t::StructBegin, 0};
    } else {
      return read_symbol;
    }
  }

  template <typename SymbolT>
  constexpr CUDF_HOST_DEVICE int32_t operator()(StateT const state_id,
                                                SymbolGroupT const match_id,
                                                SymbolT const read_symbol) const
  {
    // Number of tokens emitted on invalid lines
    constexpr int32_t num_inv_tokens = 2;

    const bool is_delimiter = match_id == static_cast<SymbolGroupT>(dfa_symbol_group_id::DELIMITER);

    // If state is either invalid or we're entering an invalid state, we discard tokens
    const bool is_part_of_invalid_line =
      (match_id != static_cast<SymbolGroupT>(dfa_symbol_group_id::ERROR) &&
       state_id == static_cast<StateT>(TT_VLD));

    // Indicates whether we transition from an invalid line to a potentially valid line
    const bool is_end_of_invalid_line = (state_id == static_cast<StateT>(TT_INV) && is_delimiter);

    int32_t const emit_count =
      is_end_of_invalid_line ? num_inv_tokens : (is_part_of_invalid_line && !is_delimiter ? 1 : 0);
    return emit_count;
  }
};

// Transition table
std::array<std::array<dfa_states, NUM_SYMBOL_GROUPS>, TT_NUM_STATES> const transition_table{
  {/* IN_STATE      ERROR   DELIM   OTHER */
   /* VALID    */ {{TT_INV, TT_VLD, TT_VLD}},
   /* INVALID  */ {{TT_INV, TT_VLD, TT_INV}}}};

// The DFA's starting state
constexpr auto start_state = static_cast<StateT>(TT_VLD);
}  // namespace token_filter

// JSON to stack operator DFA (Deterministic Finite Automata)
namespace to_stack_op {

// Type used to represent the target state in the transition table
using StateT = char;

/**
 * @brief Definition of the DFA's states
 */
enum class dfa_states : StateT {
  // The active state while outside of a string. When encountering an opening bracket or curly
  // brace, we push it onto the stack. When encountering a closing bracket or brace, we pop from the
  // stack.
  TT_OOS = 0U,

  // The active state while within a string (e.g., field name or a string value). We do not push or
  // pop from the stack while in this state.
  TT_STR,

  // The active state after encountering an escape symbol (e.g., '\'), while in the TT_STR state.
  TT_ESC,

  // Total number of states
  TT_NUM_STATES
};

// Aliases for readability of the transition table
constexpr auto TT_OOS = dfa_states::TT_OOS;
constexpr auto TT_STR = dfa_states::TT_STR;
constexpr auto TT_ESC = dfa_states::TT_ESC;

/**
 * @brief Definition of the symbol groups
 */
enum class dfa_symbol_group_id : uint8_t {
  OPENING_BRACE,     ///< Opening brace SG: {
  OPENING_BRACKET,   ///< Opening bracket SG: [
  CLOSING_BRACE,     ///< Closing brace SG: }
  CLOSING_BRACKET,   ///< Closing bracket SG: ]
  QUOTE_CHAR,        ///< Quote character SG: "
  ESCAPE_CHAR,       ///< Escape character SG: '\'
  DELIMITER_CHAR,    ///< Delimiter character SG
  OTHER_SYMBOLS,     ///< SG implicitly matching all other characters
  NUM_SYMBOL_GROUPS  ///< Total number of symbol groups
};

constexpr auto TT_NUM_STATES     = static_cast<StateT>(dfa_states::TT_NUM_STATES);
constexpr auto NUM_SYMBOL_GROUPS = static_cast<uint32_t>(dfa_symbol_group_id::NUM_SYMBOL_GROUPS);

// The DFA's starting state
constexpr auto start_state = static_cast<StateT>(TT_OOS);

template <typename SymbolT>
auto get_sgid_lut(SymbolT delim)
{
  // The i-th string representing all the characters of a symbol group
  std::array<std::vector<SymbolT>, NUM_SYMBOL_GROUPS - 1> symbol_groups{
    {{'{'}, {'['}, {'}'}, {']'}, {'"'}, {'\\'}, {delim}}};

  return symbol_groups;
}

auto get_transition_table(stack_behavior_t stack_behavior)
{
  // Transition table for the default JSON and JSON lines formats
  std::array<std::array<dfa_states, NUM_SYMBOL_GROUPS>, TT_NUM_STATES> const transition_table{
    {/* IN_STATE          {       [       }       ]       "       \      \n    OTHER */
     /* TT_OOS    */ {{TT_OOS, TT_OOS, TT_OOS, TT_OOS, TT_STR, TT_OOS, TT_OOS, TT_OOS}},
     /* TT_STR    */ {{TT_STR, TT_STR, TT_STR, TT_STR, TT_OOS, TT_ESC, TT_STR, TT_STR}},
     /* TT_ESC    */ {{TT_STR, TT_STR, TT_STR, TT_STR, TT_STR, TT_STR, TT_STR, TT_STR}}}};

  // Transition table for the JSON lines format that recovers from invalid JSON lines
  std::array<std::array<dfa_states, NUM_SYMBOL_GROUPS>, TT_NUM_STATES> const
    resetting_transition_table{
      {/* IN_STATE          {       [       }       ]       "       \      \n    OTHER */
       /* TT_OOS    */ {{TT_OOS, TT_OOS, TT_OOS, TT_OOS, TT_STR, TT_OOS, TT_OOS, TT_OOS}},
       /* TT_STR    */ {{TT_STR, TT_STR, TT_STR, TT_STR, TT_OOS, TT_ESC, TT_OOS, TT_STR}},
       /* TT_ESC    */ {{TT_STR, TT_STR, TT_STR, TT_STR, TT_STR, TT_STR, TT_OOS, TT_STR}}}};

  // Transition table specialized on the choice of whether to reset on newlines
  return (stack_behavior == stack_behavior_t::ResetOnDelimiter) ? resetting_transition_table
                                                                : transition_table;
}

auto get_translation_table(stack_behavior_t stack_behavior)
{
  // Translation table for the default JSON and JSON lines formats
  std::array<std::array<std::vector<char>, NUM_SYMBOL_GROUPS>, TT_NUM_STATES> const
    translation_table{
      {/* IN_STATE         {      [      }      ]      "      \     <delim>    OTHER */
       /* TT_OOS    */ {{{'{'}, {'['}, {'}'}, {']'}, {}, {}, {}, {}}},
       /* TT_STR    */ {{{}, {}, {}, {}, {}, {}, {}, {}}},
       /* TT_ESC    */ {{{}, {}, {}, {}, {}, {}, {}, {}}}}};

  // Translation table for the JSON lines format that recovers from invalid JSON lines
  std::array<std::array<std::vector<char>, NUM_SYMBOL_GROUPS>, TT_NUM_STATES> const
    resetting_translation_table{
      {/* IN_STATE         {      [      }      ]      "      \     <delim>    OTHER */
       /* TT_OOS    */ {{{'{'}, {'['}, {'}'}, {']'}, {}, {}, {'\n'}, {}}},
       /* TT_STR    */ {{{}, {}, {}, {}, {}, {}, {'\n'}, {}}},
       /* TT_ESC    */ {{{}, {}, {}, {}, {}, {}, {'\n'}, {}}}}};

  // Translation table specialized on the choice of whether to reset on newlines
  return stack_behavior == stack_behavior_t::ResetOnDelimiter ? resetting_translation_table
                                                              : translation_table;
}

}  // namespace to_stack_op

// JSON tokenizer pushdown automaton
namespace tokenizer_pda {

// Type used to represent the target state in the transition table
using StateT = char;

/**
 * @brief Symbol groups for the input alphabet for the pushdown automaton
 */
enum class symbol_group_id : PdaSymbolGroupIdT {
  /// Opening brace
  OPENING_BRACE,
  /// Opening bracket
  OPENING_BRACKET,
  /// Closing brace
  CLOSING_BRACE,
  /// Closing bracket
  CLOSING_BRACKET,
  /// Quote
  QUOTE,
  /// Escape
  ESCAPE,
  /// Comma
  COMMA,
  /// Colon
  COLON,
  /// Whitespace
  WHITE_SPACE,
  /// Linebreak
  LINE_BREAK,
  /// Other (any input symbol not assigned to one of the above symbol groups)
  OTHER,
  /// Total number of symbol groups amongst which to differentiate
  NUM_PDA_INPUT_SGS
};

/**
 * @brief Symbols in the stack alphabet
 */
enum class stack_symbol_group_id : PdaStackSymbolGroupIdT {
  /// Symbol representing that we're at the JSON root (nesting level 0)
  STACK_ROOT,

  /// Symbol representing that we're currently within a list object
  STACK_LIST,

  /// Symbol representing that we're currently within a struct object
  STACK_STRUCT,

  /// Total number of symbols in the stack alphabet
  NUM_STACK_SGS
};
constexpr auto NUM_PDA_INPUT_SGS =
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::NUM_PDA_INPUT_SGS);
constexpr auto NUM_STACK_SGS =
  static_cast<PdaStackSymbolGroupIdT>(stack_symbol_group_id::NUM_STACK_SGS);

/// Total number of symbol groups to differentiate amongst (stack alphabet * input alphabet)
constexpr PdaSymbolGroupIdT NUM_PDA_SGIDS = NUM_PDA_INPUT_SGS * NUM_STACK_SGS;

/// Mapping a input symbol to the symbol group id
static __constant__ PdaSymbolGroupIdT tos_sg_to_pda_sgid[] = {
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::OTHER),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::OTHER),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::OTHER),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::OTHER),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::OTHER),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::OTHER),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::OTHER),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::OTHER),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::OTHER),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::WHITE_SPACE),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::LINE_BREAK),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::OTHER),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::OTHER),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::WHITE_SPACE),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::OTHER),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::OTHER),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::OTHER),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::OTHER),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::OTHER),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::OTHER),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::OTHER),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::OTHER),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::OTHER),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::OTHER),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::OTHER),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::OTHER),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::OTHER),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::OTHER),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::OTHER),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::OTHER),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::OTHER),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::OTHER),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::WHITE_SPACE),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::OTHER),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::QUOTE),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::OTHER),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::OTHER),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::OTHER),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::OTHER),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::OTHER),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::OTHER),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::OTHER),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::OTHER),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::OTHER),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::COMMA),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::OTHER),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::OTHER),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::OTHER),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::OTHER),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::OTHER),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::OTHER),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::OTHER),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::OTHER),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::OTHER),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::OTHER),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::OTHER),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::OTHER),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::OTHER),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::COLON),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::OTHER),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::OTHER),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::OTHER),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::OTHER),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::OTHER),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::OTHER),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::OTHER),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::OTHER),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::OTHER),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::OTHER),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::OTHER),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::OTHER),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::OTHER),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::OTHER),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::OTHER),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::OTHER),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::OTHER),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::OTHER),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::OTHER),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::OTHER),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::OTHER),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::OTHER),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::OTHER),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::OTHER),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::OTHER),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::OTHER),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::OTHER),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::OTHER),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::OTHER),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::OTHER),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::OTHER),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::OTHER),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::OPENING_BRACKET),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::ESCAPE),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::CLOSING_BRACKET),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::OTHER),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::OTHER),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::OTHER),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::OTHER),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::OTHER),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::OTHER),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::OTHER),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::OTHER),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::OTHER),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::OTHER),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::OTHER),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::OTHER),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::OTHER),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::OTHER),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::OTHER),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::OTHER),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::OTHER),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::OTHER),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::OTHER),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::OTHER),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::OTHER),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::OTHER),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::OTHER),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::OTHER),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::OTHER),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::OTHER),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::OTHER),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::OTHER),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::OTHER),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::OPENING_BRACE),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::OTHER),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::CLOSING_BRACE),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::OTHER)};

/**
 * @brief Maps a (top-of-stack symbol, input symbol)-pair to a symbol group id of the deterministic
 * visibly pushdown automaton (DVPA)
 */
struct PdaSymbolToSymbolGroupId {
  SymbolT delimiter = '\n';
  template <typename SymbolT, typename StackSymbolT>
  __device__ __forceinline__ PdaSymbolGroupIdT
  operator()(thrust::tuple<SymbolT, StackSymbolT> symbol_pair) const
  {
    // The symbol read from the input
    auto symbol = thrust::get<0>(symbol_pair);

    // The stack symbol (i.e., what is on top of the stack at the time the input symbol was read)
    // I.e., whether we're reading in something within a struct, a list, or the JSON root
    auto stack_symbol = thrust::get<1>(symbol_pair);

    // The stack symbol offset: '_' is the root group (0), '[' is the list group (1), '{' is the
    // struct group (2)
    int32_t stack_idx = static_cast<PdaStackSymbolGroupIdT>(
      (stack_symbol == '_') ? stack_symbol_group_id::STACK_ROOT
                            : ((stack_symbol == '[') ? stack_symbol_group_id::STACK_LIST
                                                     : stack_symbol_group_id::STACK_STRUCT));

    // The relative symbol group id of the current input symbol
    constexpr auto pda_sgid_lookup_size =
      static_cast<int32_t>(sizeof(tos_sg_to_pda_sgid) / sizeof(tos_sg_to_pda_sgid[0]));
    // We map the delimiter character to LINE_BREAK symbol group id, and the newline character
    // to OTHER. Note that delimiter cannot be any of opening(closing) brace, bracket, quote,
    // escape, comma, colon or whitespace characters.
    auto const symbol_position =
      symbol == delimiter
        ? static_cast<int32_t>('\n')
        : (symbol == '\n' ? static_cast<int32_t>(delimiter) : static_cast<int32_t>(symbol));
    PdaSymbolGroupIdT symbol_gid =
      tos_sg_to_pda_sgid[min(symbol_position, pda_sgid_lookup_size - 1)];
    return stack_idx * static_cast<PdaSymbolGroupIdT>(symbol_group_id::NUM_PDA_INPUT_SGS) +
           symbol_gid;
  }
};

// The states defined by the pushdown automaton
enum class pda_state_t : StateT {
  // Beginning of value
  PD_BOV,
  // Beginning of array
  PD_BOA,
  // Literal or number
  PD_LON,
  // String
  PD_STR,
  // After escape char when within string
  PD_SCE,
  // After having parsed a value
  PD_PVL,
  // Before the next field name
  PD_BFN,
  // Field name
  PD_FLN,
  // After escape char when within field name
  PD_FNE,
  // After a field name inside a struct
  PD_PFN,
  // Error state (trap state)
  PD_ERR,
  // Total number of PDA states
  PD_NUM_STATES
};

enum class json_format_cfg_t {
  // Format describing regular JSON
  JSON,

  // Format describing permissive newline-delimited JSON
  // I.e., newline characters are only treteated as delimiters at the root stack level
  // E.g., this is treated as a single record:
  // {"a":
  //  123}
  JSON_LINES,

  // Format describing strict newline-delimited JSON
  // I.e., All newlines are delimiting a record, independent of the context they appear in
  JSON_LINES_STRICT,

  // Transition table for parsing newline-delimited JSON that recovers from invalid JSON lines
  // This format also follows `JSON_LINES_STRICT` behaviour
  JSON_LINES_RECOVER

};

// Aliases for readability of the transition table
constexpr auto PD_BOV = pda_state_t::PD_BOV;
constexpr auto PD_BOA = pda_state_t::PD_BOA;
constexpr auto PD_LON = pda_state_t::PD_LON;
constexpr auto PD_STR = pda_state_t::PD_STR;
constexpr auto PD_SCE = pda_state_t::PD_SCE;
constexpr auto PD_PVL = pda_state_t::PD_PVL;
constexpr auto PD_BFN = pda_state_t::PD_BFN;
constexpr auto PD_FLN = pda_state_t::PD_FLN;
constexpr auto PD_FNE = pda_state_t::PD_FNE;
constexpr auto PD_PFN = pda_state_t::PD_PFN;
constexpr auto PD_ERR = pda_state_t::PD_ERR;

constexpr auto PD_NUM_STATES = static_cast<StateT>(pda_state_t::PD_NUM_STATES);

// The starting state of the pushdown automaton
constexpr auto start_state = static_cast<StateT>(pda_state_t::PD_BOV);

/**
 * @brief Getting the transition table
 */
auto get_transition_table(json_format_cfg_t format)
{
  static_assert(static_cast<PdaStackSymbolGroupIdT>(stack_symbol_group_id::STACK_ROOT) == 0);
  static_assert(static_cast<PdaStackSymbolGroupIdT>(stack_symbol_group_id::STACK_LIST) == 1);
  static_assert(static_cast<PdaStackSymbolGroupIdT>(stack_symbol_group_id::STACK_STRUCT) == 2);

  std::array<std::array<pda_state_t, NUM_PDA_SGIDS>, PD_NUM_STATES> pda_tt;

  if (format == json_format_cfg_t::JSON || format == json_format_cfg_t::JSON_LINES) {
    // In case of newline-delimited JSON, multiple newlines are ignored, similar to whitespace.
    // Thas is, empty lines are ignored
    // PD_ANL describes the target state after a new line on an empty stack (JSON root level)
    auto const PD_ANL = (format == json_format_cfg_t::JSON) ? PD_PVL : PD_BOV;

    // First row:  empty stack         ("root" level of the JSON)
    // Second row: '[' on top of stack (we're parsing a list value)
    // Third row:  '{' on top of stack (we're parsing a struct value)
    //  {       [       }       ]       "       \       ,       :     space   newline other
    pda_tt[static_cast<StateT>(pda_state_t::PD_BOV)] = {
      PD_BOA, PD_BOA, PD_ERR, PD_ERR, PD_STR, PD_ERR, PD_ERR, PD_ERR, PD_BOV, PD_BOV, PD_LON,
      PD_BOA, PD_BOA, PD_ERR, PD_PVL, PD_STR, PD_ERR, PD_ERR, PD_ERR, PD_BOV, PD_BOV, PD_LON,
      PD_BOA, PD_BOA, PD_ERR, PD_ERR, PD_STR, PD_ERR, PD_ERR, PD_ERR, PD_BOV, PD_BOV, PD_LON};
    pda_tt[static_cast<StateT>(pda_state_t::PD_BOA)] = {
      PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_ERR,
      PD_BOA, PD_BOA, PD_ERR, PD_PVL, PD_STR, PD_ERR, PD_ERR, PD_ERR, PD_BOA, PD_BOA, PD_LON,
      PD_ERR, PD_ERR, PD_PVL, PD_ERR, PD_FLN, PD_ERR, PD_ERR, PD_ERR, PD_BOA, PD_BOA, PD_ERR};
    pda_tt[static_cast<StateT>(pda_state_t::PD_LON)] = {
      PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_PVL, PD_PVL, PD_LON,
      PD_ERR, PD_ERR, PD_ERR, PD_PVL, PD_ERR, PD_ERR, PD_BOV, PD_ERR, PD_PVL, PD_PVL, PD_LON,
      PD_ERR, PD_ERR, PD_PVL, PD_ERR, PD_ERR, PD_ERR, PD_BFN, PD_ERR, PD_PVL, PD_PVL, PD_LON};
    pda_tt[static_cast<StateT>(pda_state_t::PD_STR)] = {
      PD_STR, PD_STR, PD_STR, PD_STR, PD_PVL, PD_SCE, PD_STR, PD_STR, PD_STR, PD_STR, PD_STR,
      PD_STR, PD_STR, PD_STR, PD_STR, PD_PVL, PD_SCE, PD_STR, PD_STR, PD_STR, PD_STR, PD_STR,
      PD_STR, PD_STR, PD_STR, PD_STR, PD_PVL, PD_SCE, PD_STR, PD_STR, PD_STR, PD_STR, PD_STR};
    pda_tt[static_cast<StateT>(pda_state_t::PD_SCE)] = {
      PD_STR, PD_STR, PD_STR, PD_STR, PD_STR, PD_STR, PD_STR, PD_STR, PD_STR, PD_STR, PD_STR,
      PD_STR, PD_STR, PD_STR, PD_STR, PD_STR, PD_STR, PD_STR, PD_STR, PD_STR, PD_STR, PD_STR,
      PD_STR, PD_STR, PD_STR, PD_STR, PD_STR, PD_STR, PD_STR, PD_STR, PD_STR, PD_STR, PD_STR};
    pda_tt[static_cast<StateT>(pda_state_t::PD_PVL)] = {
      PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_PVL, PD_ANL, PD_ERR,
      PD_ERR, PD_ERR, PD_ERR, PD_PVL, PD_ERR, PD_ERR, PD_BOV, PD_ERR, PD_PVL, PD_PVL, PD_ERR,
      PD_ERR, PD_ERR, PD_PVL, PD_ERR, PD_ERR, PD_ERR, PD_BFN, PD_ERR, PD_PVL, PD_PVL, PD_ERR};
    pda_tt[static_cast<StateT>(pda_state_t::PD_BFN)] = {
      PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_ERR,
      PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_ERR,
      PD_ERR, PD_ERR, PD_PVL, PD_ERR, PD_FLN, PD_ERR, PD_ERR, PD_ERR, PD_BFN, PD_BFN, PD_ERR};
    pda_tt[static_cast<StateT>(pda_state_t::PD_FLN)] = {
      PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_ERR,
      PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_ERR,
      PD_FLN, PD_FLN, PD_FLN, PD_FLN, PD_PFN, PD_FNE, PD_FLN, PD_FLN, PD_FLN, PD_FLN, PD_FLN};
    pda_tt[static_cast<StateT>(pda_state_t::PD_FNE)] = {
      PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_ERR,
      PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_ERR,
      PD_FLN, PD_FLN, PD_FLN, PD_FLN, PD_FLN, PD_FLN, PD_FLN, PD_FLN, PD_FLN, PD_FLN, PD_FLN};
    pda_tt[static_cast<StateT>(pda_state_t::PD_PFN)] = {
      PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_ERR,
      PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_ERR,
      PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_BOV, PD_PFN, PD_PFN, PD_ERR};
    pda_tt[static_cast<StateT>(pda_state_t::PD_ERR)] = {
      PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_ERR,
      PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_ERR,
      PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_ERR};
  }
  // Transition table for strict JSON lines (including recovery)
  // Newlines are treated as record delimiters
  else {
    // In case of newline-delimited JSON, multiple newlines are ignored, similar to whitespace.
    // Thas is, empty lines are ignored
    // PD_ANL describes the target state after a new line after encountering error state
    auto const PD_ANL = (format == json_format_cfg_t::JSON_LINES_RECOVER) ? PD_BOV : PD_ERR;

    // Target state after having parsed the first JSON value on a JSON line
    // Spark has the special need to ignore everything that comes after the first JSON object
    // on a JSON line instead of marking those as invalid
    auto const PD_AFS = (format == json_format_cfg_t::JSON_LINES_RECOVER) ? PD_PVL : PD_ERR;

    // First row:  empty stack         ("root" level of the JSON)
    // Second row: '[' on top of stack (we're parsing a list value)
    // Third row:  '{' on top of stack (we're parsing a struct value)
    //  {       [       }       ]       "       \       ,       :     space   newline other
    pda_tt[static_cast<StateT>(pda_state_t::PD_BOV)] = {
      PD_BOA, PD_BOA, PD_ERR, PD_ERR, PD_STR, PD_ERR, PD_ERR, PD_ERR, PD_BOV, PD_BOV, PD_LON,
      PD_BOA, PD_BOA, PD_ERR, PD_PVL, PD_STR, PD_ERR, PD_ERR, PD_ERR, PD_BOV, PD_BOV, PD_LON,
      PD_BOA, PD_BOA, PD_ERR, PD_ERR, PD_STR, PD_ERR, PD_ERR, PD_ERR, PD_BOV, PD_BOV, PD_LON};
    pda_tt[static_cast<StateT>(pda_state_t::PD_BOA)] = {
      PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_BOV, PD_ERR,
      PD_BOA, PD_BOA, PD_ERR, PD_PVL, PD_STR, PD_ERR, PD_ERR, PD_ERR, PD_BOA, PD_BOV, PD_LON,
      PD_ERR, PD_ERR, PD_PVL, PD_ERR, PD_FLN, PD_ERR, PD_ERR, PD_ERR, PD_BOA, PD_BOV, PD_ERR};
    pda_tt[static_cast<StateT>(pda_state_t::PD_LON)] = {
      PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_PVL, PD_BOV, PD_LON,
      PD_ERR, PD_ERR, PD_ERR, PD_PVL, PD_ERR, PD_ERR, PD_BOV, PD_ERR, PD_PVL, PD_BOV, PD_LON,
      PD_ERR, PD_ERR, PD_PVL, PD_ERR, PD_ERR, PD_ERR, PD_BFN, PD_ERR, PD_PVL, PD_BOV, PD_LON};
    pda_tt[static_cast<StateT>(pda_state_t::PD_STR)] = {
      PD_STR, PD_STR, PD_STR, PD_STR, PD_PVL, PD_SCE, PD_STR, PD_STR, PD_STR, PD_BOV, PD_STR,
      PD_STR, PD_STR, PD_STR, PD_STR, PD_PVL, PD_SCE, PD_STR, PD_STR, PD_STR, PD_BOV, PD_STR,
      PD_STR, PD_STR, PD_STR, PD_STR, PD_PVL, PD_SCE, PD_STR, PD_STR, PD_STR, PD_BOV, PD_STR};
    pda_tt[static_cast<StateT>(pda_state_t::PD_SCE)] = {
      PD_STR, PD_STR, PD_STR, PD_STR, PD_STR, PD_STR, PD_STR, PD_STR, PD_STR, PD_BOV, PD_STR,
      PD_STR, PD_STR, PD_STR, PD_STR, PD_STR, PD_STR, PD_STR, PD_STR, PD_STR, PD_BOV, PD_STR,
      PD_STR, PD_STR, PD_STR, PD_STR, PD_STR, PD_STR, PD_STR, PD_STR, PD_STR, PD_BOV, PD_STR};
    pda_tt[static_cast<StateT>(pda_state_t::PD_PVL)] = {
      PD_AFS, PD_AFS, PD_AFS, PD_AFS, PD_AFS, PD_AFS, PD_AFS, PD_AFS, PD_PVL, PD_BOV, PD_AFS,
      PD_ERR, PD_ERR, PD_ERR, PD_PVL, PD_ERR, PD_ERR, PD_BOV, PD_ERR, PD_PVL, PD_BOV, PD_ERR,
      PD_ERR, PD_ERR, PD_PVL, PD_ERR, PD_ERR, PD_ERR, PD_BFN, PD_ERR, PD_PVL, PD_BOV, PD_ERR};
    pda_tt[static_cast<StateT>(pda_state_t::PD_BFN)] = {
      PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_BOV, PD_ERR,
      PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_BOV, PD_ERR,
      PD_ERR, PD_ERR, PD_PVL, PD_ERR, PD_FLN, PD_ERR, PD_ERR, PD_ERR, PD_BFN, PD_BOV, PD_ERR};
    pda_tt[static_cast<StateT>(pda_state_t::PD_FLN)] = {
      PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_BOV, PD_ERR,
      PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_BOV, PD_ERR,
      PD_FLN, PD_FLN, PD_FLN, PD_FLN, PD_PFN, PD_FNE, PD_FLN, PD_FLN, PD_FLN, PD_BOV, PD_FLN};
    pda_tt[static_cast<StateT>(pda_state_t::PD_FNE)] = {
      PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_BOV, PD_ERR,
      PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_BOV, PD_ERR,
      PD_FLN, PD_FLN, PD_FLN, PD_FLN, PD_FLN, PD_FLN, PD_FLN, PD_FLN, PD_FLN, PD_BOV, PD_FLN};
    pda_tt[static_cast<StateT>(pda_state_t::PD_PFN)] = {
      PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_BOV, PD_ERR,
      PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_BOV, PD_ERR,
      PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_BOV, PD_PFN, PD_BOV, PD_ERR};
    pda_tt[static_cast<StateT>(pda_state_t::PD_ERR)] = {
      PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_ANL, PD_ERR,
      PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_ANL, PD_ERR,
      PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_ANL, PD_ERR};
  }
  return pda_tt;
}

/**
 * @brief Getting the translation table
 * @param recover_from_error Whether or not the tokenizer should recover from invalid lines. If
 * `recover_from_error` is true, invalid JSON lines end with the token sequence (`ErrorBegin`,
 * `LineEn`) and incomplete JSON lines (e.g., `{"a":123\n`) are treated as invalid lines.
 */
auto get_translation_table(bool recover_from_error)
{
  constexpr auto StructBegin       = token_t::StructBegin;
  constexpr auto StructEnd         = token_t::StructEnd;
  constexpr auto ListBegin         = token_t::ListBegin;
  constexpr auto ListEnd           = token_t::ListEnd;
  constexpr auto StructMemberBegin = token_t::StructMemberBegin;
  constexpr auto StructMemberEnd   = token_t::StructMemberEnd;
  constexpr auto FieldNameBegin    = token_t::FieldNameBegin;
  constexpr auto FieldNameEnd      = token_t::FieldNameEnd;
  constexpr auto StringBegin       = token_t::StringBegin;
  constexpr auto StringEnd         = token_t::StringEnd;
  constexpr auto ValueBegin        = token_t::ValueBegin;
  constexpr auto ValueEnd          = token_t::ValueEnd;
  constexpr auto ErrorBegin        = token_t::ErrorBegin;

  /**
   * @brief Instead of specifying the verbose translation tables twice (i.e., once when
   * `recover_from_error` is true and once when it is false), we use `nl_tokens` to specialize the
   * translation table where it differs depending on the `recover_from_error` option. If and only if
   * `recover_from_error` is true, `recovering_tokens` are returned along with a token_t::LineEnd
   * token, otherwise `regular_tokens` is returned.
   */
  auto nl_tokens = [recover_from_error](std::vector<char> regular_tokens,
                                        std::vector<char> recovering_tokens) {
    if (recover_from_error) {
      recovering_tokens.push_back(token_t::LineEnd);
      return recovering_tokens;
    }
    return regular_tokens;
  };

  /**
   * @brief Helper function that returns `recovering_tokens` if `recover_from_error` is true and
   * returns `regular_tokens` otherwise. This is used to ignore excess characters after the first
   * value in the case of JSON lines that recover from invalid lines, as Spark ignores any excess
   * characters that follow the first record on a JSON line.
   */
  auto alt_tokens = [recover_from_error](std::vector<char> regular_tokens,
                                         std::vector<char> recovering_tokens) {
    if (recover_from_error) { return recovering_tokens; }
    return regular_tokens;
  };

  std::array<std::array<std::vector<char>, NUM_PDA_SGIDS>, PD_NUM_STATES> pda_tlt;
  pda_tlt[static_cast<StateT>(pda_state_t::PD_BOV)] = {{                    /*ROOT*/
                                                        {StructBegin},      // OPENING_BRACE
                                                        {ListBegin},        // OPENING_BRACKET
                                                        {ErrorBegin},       // CLOSING_BRACE
                                                        {ErrorBegin},       // CLOSING_BRACKET
                                                        {StringBegin},      // QUOTE
                                                        {ErrorBegin},       // ESCAPE
                                                        {ErrorBegin},       // COMMA
                                                        {ErrorBegin},       // COLON
                                                        {},                 // WHITE_SPACE
                                                        nl_tokens({}, {}),  // LINE_BREAK
                                                        {ValueBegin},       // OTHER
                                                        /*LIST*/
                                                        {StructBegin},  // OPENING_BRACE
                                                        {ListBegin},    // OPENING_BRACKET
                                                        {ErrorBegin},   // CLOSING_BRACE
                                                        {ListEnd},      // CLOSING_BRACKET
                                                        {StringBegin},  // QUOTE
                                                        {ErrorBegin},   // ESCAPE
                                                        {ErrorBegin},   // COMMA
                                                        {ErrorBegin},   // COLON
                                                        {},             // WHITE_SPACE
                                                        nl_tokens({}, {ErrorBegin}),  // LINE_BREAK
                                                        {ValueBegin},                 // OTHER
                                                        /*STRUCT*/
                                                        {StructBegin},  // OPENING_BRACE
                                                        {ListBegin},    // OPENING_BRACKET
                                                        {ErrorBegin},   // CLOSING_BRACE
                                                        {ErrorBegin},   // CLOSING_BRACKET
                                                        {StringBegin},  // QUOTE
                                                        {ErrorBegin},   // ESCAPE
                                                        {ErrorBegin},   // COMMA
                                                        {ErrorBegin},   // COLON
                                                        {},             // WHITE_SPACE
                                                        nl_tokens({}, {ErrorBegin}),  // LINE_BREAK
                                                        {ValueBegin}}};               // OTHER
  pda_tlt[static_cast<StateT>(pda_state_t::PD_BOA)] = {
    {                                        /*ROOT*/
     {ErrorBegin},                           // OPENING_BRACE
     {ErrorBegin},                           // OPENING_BRACKET
     {ErrorBegin},                           // CLOSING_BRACE
     {ErrorBegin},                           // CLOSING_BRACKET
     {ErrorBegin},                           // QUOTE
     {ErrorBegin},                           // ESCAPE
     {ErrorBegin},                           // COMMA
     {ErrorBegin},                           // COLON
     {ErrorBegin},                           // WHITE_SPACE
     nl_tokens({ErrorBegin}, {ErrorBegin}),  // LINE_BREAK
     {ErrorBegin},                           // OTHER
     /*LIST*/
     {StructBegin},                // OPENING_BRACE
     {ListBegin},                  // OPENING_BRACKET
     {ErrorBegin},                 // CLOSING_BRACE
     {ListEnd},                    // CLOSING_BRACKET
     {StringBegin},                // QUOTE
     {ErrorBegin},                 // ESCAPE
     {ErrorBegin},                 // COMMA
     {ErrorBegin},                 // COLON
     {},                           // WHITE_SPACE
     nl_tokens({}, {ErrorBegin}),  // LINE_BREAK
     {ValueBegin},                 // OTHER
     /*STRUCT*/
     {ErrorBegin},                         // OPENING_BRACE
     {ErrorBegin},                         // OPENING_BRACKET
     {StructEnd},                          // CLOSING_BRACE
     {ErrorBegin},                         // CLOSING_BRACKET
     {StructMemberBegin, FieldNameBegin},  // QUOTE
     {ErrorBegin},                         // ESCAPE
     {ErrorBegin},                         // COMMA
     {ErrorBegin},                         // COLON
     {},                                   // WHITE_SPACE
     nl_tokens({}, {ErrorBegin}),          // LINE_BREAK
     {ErrorBegin}}};                       // OTHER
  pda_tlt[static_cast<StateT>(pda_state_t::PD_LON)] = {
    {                                      /*ROOT*/
     {ErrorBegin},                         // OPENING_BRACE
     {ErrorBegin},                         // OPENING_BRACKET
     {ErrorBegin},                         // CLOSING_BRACE
     {ErrorBegin},                         // CLOSING_BRACKET
     {ErrorBegin},                         // QUOTE
     {ErrorBegin},                         // ESCAPE
     {ErrorBegin},                         // COMMA
     {ErrorBegin},                         // COLON
     {ValueEnd},                           // WHITE_SPACE
     nl_tokens({ValueEnd}, {ErrorBegin}),  // LINE_BREAK
     {},                                   // OTHER
     /*LIST*/
     {ErrorBegin},                         // OPENING_BRACE
     {ErrorBegin},                         // OPENING_BRACKET
     {ErrorBegin},                         // CLOSING_BRACE
     {ValueEnd, ListEnd},                  // CLOSING_BRACKET
     {ErrorBegin},                         // QUOTE
     {ErrorBegin},                         // ESCAPE
     {ValueEnd},                           // COMMA
     {ErrorBegin},                         // COLON
     {ValueEnd},                           // WHITE_SPACE
     nl_tokens({ValueEnd}, {ErrorBegin}),  // LINE_BREAK
     {},                                   // OTHER
     /*STRUCT*/
     {ErrorBegin},                            // OPENING_BRACE
     {ErrorBegin},                            // OPENING_BRACKET
     {ValueEnd, StructMemberEnd, StructEnd},  // CLOSING_BRACE
     {ErrorBegin},                            // CLOSING_BRACKET
     {ErrorBegin},                            // QUOTE
     {ErrorBegin},                            // ESCAPE
     {ValueEnd, StructMemberEnd},             // COMMA
     {ErrorBegin},                            // COLON
     {ValueEnd},                              // WHITE_SPACE
     nl_tokens({ValueEnd}, {ErrorBegin}),     // LINE_BREAK
     {}}};                                    // OTHER

  pda_tlt[static_cast<StateT>(pda_state_t::PD_STR)] = {{              /*ROOT*/
                                                        {},           // OPENING_BRACE
                                                        {},           // OPENING_BRACKET
                                                        {},           // CLOSING_BRACE
                                                        {},           // CLOSING_BRACKET
                                                        {StringEnd},  // QUOTE
                                                        {},           // ESCAPE
                                                        {},           // COMMA
                                                        {},           // COLON
                                                        {},           // WHITE_SPACE
                                                        nl_tokens({}, {ErrorBegin}),  // LINE_BREAK
                                                        {},                           // OTHER
                                                        /*LIST*/
                                                        {},           // OPENING_BRACE
                                                        {},           // OPENING_BRACKET
                                                        {},           // CLOSING_BRACE
                                                        {},           // CLOSING_BRACKET
                                                        {StringEnd},  // QUOTE
                                                        {},           // ESCAPE
                                                        {},           // COMMA
                                                        {},           // COLON
                                                        {},           // WHITE_SPACE
                                                        nl_tokens({}, {ErrorBegin}),  // LINE_BREAK
                                                        {},                           // OTHER
                                                        /*STRUCT*/
                                                        {},           // OPENING_BRACE
                                                        {},           // OPENING_BRACKET
                                                        {},           // CLOSING_BRACE
                                                        {},           // CLOSING_BRACKET
                                                        {StringEnd},  // QUOTE
                                                        {},           // ESCAPE
                                                        {},           // COMMA
                                                        {},           // COLON
                                                        {},           // WHITE_SPACE
                                                        nl_tokens({}, {ErrorBegin}),  // LINE_BREAK
                                                        {}}};                         // OTHER

  pda_tlt[static_cast<StateT>(pda_state_t::PD_SCE)] = {{     /*ROOT*/
                                                        {},  // OPENING_BRACE
                                                        {},  // OPENING_BRACKET
                                                        {},  // CLOSING_BRACE
                                                        {},  // CLOSING_BRACKET
                                                        {},  // QUOTE
                                                        {},  // ESCAPE
                                                        {},  // COMMA
                                                        {},  // COLON
                                                        {},  // WHITE_SPACE
                                                        nl_tokens({}, {ErrorBegin}),  // LINE_BREAK
                                                        {},                           // OTHER
                                                        /*LIST*/
                                                        {},  // OPENING_BRACE
                                                        {},  // OPENING_BRACKET
                                                        {},  // CLOSING_BRACE
                                                        {},  // CLOSING_BRACKET
                                                        {},  // QUOTE
                                                        {},  // ESCAPE
                                                        {},  // COMMA
                                                        {},  // COLON
                                                        {},  // WHITE_SPACE
                                                        nl_tokens({}, {ErrorBegin}),  // LINE_BREAK
                                                        {},                           // OTHER
                                                        /*STRUCT*/
                                                        {},  // OPENING_BRACE
                                                        {},  // OPENING_BRACKET
                                                        {},  // CLOSING_BRACE
                                                        {},  // CLOSING_BRACKET
                                                        {},  // QUOTE
                                                        {},  // ESCAPE
                                                        {},  // COMMA
                                                        {},  // COLON
                                                        {},  // WHITE_SPACE
                                                        nl_tokens({}, {ErrorBegin}),  // LINE_BREAK
                                                        {}}};                         // OTHER

  pda_tlt[static_cast<StateT>(pda_state_t::PD_PVL)] = {
    {                                 /*ROOT*/
     {alt_tokens({ErrorBegin}, {})},  // OPENING_BRACE
     {alt_tokens({ErrorBegin}, {})},  // OPENING_BRACKET
     {alt_tokens({ErrorBegin}, {})},  // CLOSING_BRACE
     {alt_tokens({ErrorBegin}, {})},  // CLOSING_BRACKET
     {alt_tokens({ErrorBegin}, {})},  // QUOTE
     {alt_tokens({ErrorBegin}, {})},  // ESCAPE
     {alt_tokens({ErrorBegin}, {})},  // COMMA
     {alt_tokens({ErrorBegin}, {})},  // COLON
     {},                              // WHITE_SPACE
     nl_tokens({}, {}),               // LINE_BREAK
     {alt_tokens({ErrorBegin}, {})},  // OTHER
     /*LIST*/
     {ErrorBegin},                 // OPENING_BRACE
     {ErrorBegin},                 // OPENING_BRACKET
     {ErrorBegin},                 // CLOSING_BRACE
     {ListEnd},                    // CLOSING_BRACKET
     {ErrorBegin},                 // QUOTE
     {ErrorBegin},                 // ESCAPE
     {},                           // COMMA
     {ErrorBegin},                 // COLON
     {},                           // WHITE_SPACE
     nl_tokens({}, {ErrorBegin}),  // LINE_BREAK
     {ErrorBegin},                 // OTHER
     /*STRUCT*/
     {ErrorBegin},                  // OPENING_BRACE
     {ErrorBegin},                  // OPENING_BRACKET
     {StructMemberEnd, StructEnd},  // CLOSING_BRACE
     {ErrorBegin},                  // CLOSING_BRACKET
     {ErrorBegin},                  // QUOTE
     {ErrorBegin},                  // ESCAPE
     {StructMemberEnd},             // COMMA
     {ErrorBegin},                  // COLON
     {},                            // WHITE_SPACE
     nl_tokens({}, {ErrorBegin}),   // LINE_BREAK
     {ErrorBegin}}};                // OTHER

  pda_tlt[static_cast<StateT>(pda_state_t::PD_BFN)] = {
    {                                        /*ROOT*/
     {ErrorBegin},                           // OPENING_BRACE
     {ErrorBegin},                           // OPENING_BRACKET
     {ErrorBegin},                           // CLOSING_BRACE
     {ErrorBegin},                           // CLOSING_BRACKET
     {ErrorBegin},                           // QUOTE
     {ErrorBegin},                           // ESCAPE
     {ErrorBegin},                           // COMMA
     {ErrorBegin},                           // COLON
     {ErrorBegin},                           // WHITE_SPACE
     nl_tokens({ErrorBegin}, {ErrorBegin}),  // LINE_BREAK
     {ErrorBegin},                           // OTHER
     /*LIST*/
     {ErrorBegin},                           // OPENING_BRACE
     {ErrorBegin},                           // OPENING_BRACKET
     {ErrorBegin},                           // CLOSING_BRACE
     {ErrorBegin},                           // CLOSING_BRACKET
     {ErrorBegin},                           // QUOTE
     {ErrorBegin},                           // ESCAPE
     {ErrorBegin},                           // COMMA
     {ErrorBegin},                           // COLON
     {ErrorBegin},                           // WHITE_SPACE
     nl_tokens({ErrorBegin}, {ErrorBegin}),  // LINE_BREAK
     {ErrorBegin},                           // OTHER
     /*STRUCT*/
     {ErrorBegin},                         // OPENING_BRACE
     {ErrorBegin},                         // OPENING_BRACKET
     {StructEnd},                          // CLOSING_BRACE
     {ErrorBegin},                         // CLOSING_BRACKET
     {StructMemberBegin, FieldNameBegin},  // QUOTE
     {ErrorBegin},                         // ESCAPE
     {ErrorBegin},                         // COMMA
     {ErrorBegin},                         // COLON
     {},                                   // WHITE_SPACE
     nl_tokens({}, {ErrorBegin}),          // LINE_BREAK
     {ErrorBegin}}};                       // OTHER

  pda_tlt[static_cast<StateT>(pda_state_t::PD_FLN)] = {
    {                                        /*ROOT*/
     {ErrorBegin},                           // OPENING_BRACE
     {ErrorBegin},                           // OPENING_BRACKET
     {ErrorBegin},                           // CLOSING_BRACE
     {ErrorBegin},                           // CLOSING_BRACKET
     {ErrorBegin},                           // QUOTE
     {ErrorBegin},                           // ESCAPE
     {ErrorBegin},                           // COMMA
     {ErrorBegin},                           // COLON
     {ErrorBegin},                           // WHITE_SPACE
     nl_tokens({ErrorBegin}, {ErrorBegin}),  // LINE_BREAK
     {ErrorBegin},                           // OTHER
     /*LIST*/
     {ErrorBegin},                           // OPENING_BRACE
     {ErrorBegin},                           // OPENING_BRACKET
     {ErrorBegin},                           // CLOSING_BRACE
     {ErrorBegin},                           // CLOSING_BRACKET
     {ErrorBegin},                           // QUOTE
     {ErrorBegin},                           // ESCAPE
     {ErrorBegin},                           // COMMA
     {ErrorBegin},                           // COLON
     {ErrorBegin},                           // WHITE_SPACE
     nl_tokens({ErrorBegin}, {ErrorBegin}),  // LINE_BREAK
     {ErrorBegin},                           // OTHER
     /*STRUCT*/
     {},                           // OPENING_BRACE
     {},                           // OPENING_BRACKET
     {},                           // CLOSING_BRACE
     {},                           // CLOSING_BRACKET
     {FieldNameEnd},               // QUOTE
     {},                           // ESCAPE
     {},                           // COMMA
     {},                           // COLON
     {},                           // WHITE_SPACE
     nl_tokens({}, {ErrorBegin}),  // LINE_BREAK
     {}}};                         // OTHER

  pda_tlt[static_cast<StateT>(pda_state_t::PD_FNE)] = {
    {                                        /*ROOT*/
     {ErrorBegin},                           // OPENING_BRACE
     {ErrorBegin},                           // OPENING_BRACKET
     {ErrorBegin},                           // CLOSING_BRACE
     {ErrorBegin},                           // CLOSING_BRACKET
     {ErrorBegin},                           // QUOTE
     {ErrorBegin},                           // ESCAPE
     {ErrorBegin},                           // COMMA
     {ErrorBegin},                           // COLON
     {ErrorBegin},                           // WHITE_SPACE
     nl_tokens({ErrorBegin}, {ErrorBegin}),  // LINE_BREAK
     {ErrorBegin},                           // OTHER
     /*LIST*/
     {ErrorBegin},                           // OPENING_BRACE
     {ErrorBegin},                           // OPENING_BRACKET
     {ErrorBegin},                           // CLOSING_BRACE
     {ErrorBegin},                           // CLOSING_BRACKET
     {ErrorBegin},                           // QUOTE
     {ErrorBegin},                           // ESCAPE
     {ErrorBegin},                           // COMMA
     {ErrorBegin},                           // COLON
     {ErrorBegin},                           // WHITE_SPACE
     nl_tokens({ErrorBegin}, {ErrorBegin}),  // LINE_BREAK
     {ErrorBegin},                           // OTHER
     /*STRUCT*/
     {},                           // OPENING_BRACE
     {},                           // OPENING_BRACKET
     {},                           // CLOSING_BRACE
     {},                           // CLOSING_BRACKET
     {},                           // QUOTE
     {},                           // ESCAPE
     {},                           // COMMA
     {},                           // COLON
     {},                           // WHITE_SPACE
     nl_tokens({}, {ErrorBegin}),  // LINE_BREAK
     {}}};                         // OTHER

  pda_tlt[static_cast<StateT>(pda_state_t::PD_PFN)] = {
    {                                        /*ROOT*/
     {ErrorBegin},                           // OPENING_BRACE
     {ErrorBegin},                           // OPENING_BRACKET
     {ErrorBegin},                           // CLOSING_BRACE
     {ErrorBegin},                           // CLOSING_BRACKET
     {ErrorBegin},                           // QUOTE
     {ErrorBegin},                           // ESCAPE
     {ErrorBegin},                           // COMMA
     {ErrorBegin},                           // COLON
     {ErrorBegin},                           // WHITE_SPACE
     nl_tokens({ErrorBegin}, {ErrorBegin}),  // LINE_BREAK
     {ErrorBegin},                           // OTHER
     /*LIST*/
     {ErrorBegin},                           // OPENING_BRACE
     {ErrorBegin},                           // OPENING_BRACKET
     {ErrorBegin},                           // CLOSING_BRACE
     {ErrorBegin},                           // CLOSING_BRACKET
     {ErrorBegin},                           // QUOTE
     {ErrorBegin},                           // ESCAPE
     {ErrorBegin},                           // COMMA
     {ErrorBegin},                           // COLON
     {ErrorBegin},                           // WHITE_SPACE
     nl_tokens({ErrorBegin}, {ErrorBegin}),  // LINE_BREAK
     {ErrorBegin},                           // OTHER
     /*STRUCT*/
     {ErrorBegin},                 // OPENING_BRACE
     {ErrorBegin},                 // OPENING_BRACKET
     {ErrorBegin},                 // CLOSING_BRACE
     {ErrorBegin},                 // CLOSING_BRACKET
     {ErrorBegin},                 // QUOTE
     {ErrorBegin},                 // ESCAPE
     {ErrorBegin},                 // COMMA
     {},                           // COLON
     {},                           // WHITE_SPACE
     nl_tokens({}, {ErrorBegin}),  // LINE_BREAK
     {ErrorBegin}}};               // OTHER

  pda_tlt[static_cast<StateT>(pda_state_t::PD_ERR)] = {{                    /*ROOT*/
                                                        {},                 // OPENING_BRACE
                                                        {},                 // OPENING_BRACKET
                                                        {},                 // CLOSING_BRACE
                                                        {},                 // CLOSING_BRACKET
                                                        {},                 // QUOTE
                                                        {},                 // ESCAPE
                                                        {},                 // COMMA
                                                        {},                 // COLON
                                                        {},                 // WHITE_SPACE
                                                        nl_tokens({}, {}),  // LINE_BREAK
                                                        {},                 // OTHER
                                                        /*LIST*/
                                                        {},                 // OPENING_BRACE
                                                        {},                 // OPENING_BRACKET
                                                        {},                 // CLOSING_BRACE
                                                        {},                 // CLOSING_BRACKET
                                                        {},                 // QUOTE
                                                        {},                 // ESCAPE
                                                        {},                 // COMMA
                                                        {},                 // COLON
                                                        {},                 // WHITE_SPACE
                                                        nl_tokens({}, {}),  // LINE_BREAK
                                                        {},                 // OTHER
                                                        /*STRUCT*/
                                                        {},                 // OPENING_BRACE
                                                        {},                 // OPENING_BRACKET
                                                        {},                 // CLOSING_BRACE
                                                        {},                 // CLOSING_BRACKET
                                                        {},                 // QUOTE
                                                        {},                 // ESCAPE
                                                        {},                 // COMMA
                                                        {},                 // COLON
                                                        {},                 // WHITE_SPACE
                                                        nl_tokens({}, {}),  // LINE_BREAK
                                                        {}}};               // OTHER
  return pda_tlt;
}

}  // namespace tokenizer_pda

/**
 * @brief Function object used to filter for brackets and braces that represent push and pop
 * operations
 */
struct JSONToStackOp {
  template <typename StackSymbolT>
  constexpr CUDF_HOST_DEVICE fst::stack_op_type operator()(StackSymbolT const& stack_symbol) const
  {
    switch (stack_symbol) {
      case '{':
      case '[': return fst::stack_op_type::PUSH;
      case '}':
      case ']': return fst::stack_op_type::POP;
      default: return fst::stack_op_type::READ;
    }
  }
};

/**
 * @brief Function object used to filter for brackets and braces that represent push and pop
 * operations
 */
struct JSONWithRecoveryToStackOp {
  template <typename StackSymbolT>
  constexpr CUDF_HOST_DEVICE fst::stack_op_type operator()(StackSymbolT const& stack_symbol) const
  {
    switch (stack_symbol) {
      case '{':
      case '[': return fst::stack_op_type::PUSH;
      case '}':
      case ']': return fst::stack_op_type::POP;
      case '\n': return fst::stack_op_type::RESET;
      default: return fst::stack_op_type::READ;
    }
  }
};

void json_column::null_fill(row_offset_t up_to_row_offset)
{
  // Fill all the rows up to up_to_row_offset with "empty"/null rows
  validity.resize(word_index(up_to_row_offset) + 1);
  std::fill_n(std::back_inserter(string_offsets),
              up_to_row_offset - string_offsets.size(),
              (string_offsets.size() > 0) ? string_offsets.back() : 0);
  std::fill_n(std::back_inserter(string_lengths), up_to_row_offset - string_lengths.size(), 0);
  std::fill_n(std::back_inserter(child_offsets),
              up_to_row_offset + 1 - child_offsets.size(),
              (child_offsets.size() > 0) ? child_offsets.back() : 0);
  current_offset = up_to_row_offset;
}

void json_column::level_child_cols_recursively(row_offset_t min_row_count)
{
  // Fill this columns with nulls up to the given row count
  null_fill(min_row_count);

  // If this is a struct column, we need to level all its child columns
  if (type == json_col_t::StructColumn) {
    for (auto it = std::begin(child_columns); it != std::end(child_columns); it++) {
      it->second.level_child_cols_recursively(min_row_count);
    }
  }
  // If this is a list column, we need to make sure that its child column levels its children
  else if (type == json_col_t::ListColumn) {
    auto it = std::begin(child_columns);
    // Make that child column fill its child columns up to its own row count
    if (it != std::end(child_columns)) {
      it->second.level_child_cols_recursively(it->second.current_offset);
    }
  }
};

void json_column::append_row(uint32_t row_index,
                             json_col_t row_type,
                             uint32_t string_offset,
                             uint32_t string_end,
                             uint32_t child_count)
{
  // If, thus far, the column's type couldn't be inferred, we infer it to the given type
  if (type == json_col_t::Unknown) {
    type = row_type;
  }
  // If, at some point within a column, we encounter a nested type (list or struct),
  // we change that column's type to that respective nested type and invalidate all previous rows
  else if (type == json_col_t::StringColumn &&
           (row_type == json_col_t::ListColumn || row_type == json_col_t::StructColumn)) {
    // Change the column type
    type = row_type;

    // Invalidate all previous entries, as they were _not_ of the nested type to which we just
    // converted
    std::fill_n(validity.begin(), validity.size(), 0);
    valid_count = 0U;
  }
  // If this is a nested column but we're trying to insert either (a) a list node into a struct
  // column or (b) a struct node into a list column, we fail
  CUDF_EXPECTS(not((type == json_col_t::ListColumn and row_type == json_col_t::StructColumn) or
                   (type == json_col_t::StructColumn and row_type == json_col_t::ListColumn)),
               "A mix of lists and structs within the same column is not supported");

  // We shouldn't run into this, as we shouldn't be asked to append an "unknown" row type
  CUDF_EXPECTS(type != json_col_t::Unknown, "Encountered invalid JSON token sequence");

  // Fill all the omitted rows with "empty"/null rows (if needed)
  null_fill(row_index);

  // Table listing what we intend to use for a given column type and row type combination
  // col type | row type  => {valid, FAIL, null}
  // -----------------------------------------------
  // List     | List      => valid
  // List     | Struct    => FAIL
  // List     | String    => null
  // Struct   | List      => FAIL
  // Struct   | Struct    => valid
  // Struct   | String    => null
  // String   | List      => valid (we switch col type to list, null'ing all previous rows)
  // String   | Struct    => valid (we switch col type to list, null'ing all previous rows)
  // String   | String    => valid
  bool const is_valid = (type == row_type);
  if (static_cast<size_type>(validity.size()) < word_index(current_offset)) validity.push_back({});
  if (is_valid) { set_bit_unsafe(&validity.back(), intra_word_index(current_offset)); }
  valid_count += (is_valid) ? 1U : 0U;
  string_offsets.push_back(string_offset);
  string_lengths.push_back(string_end - string_offset);
  child_offsets.push_back((child_offsets.size() > 0) ? child_offsets.back() + child_count : 0);
  current_offset++;
};

namespace detail {

void get_stack_context(device_span<SymbolT const> json_in,
                       SymbolT* d_top_of_stack,
                       stack_behavior_t stack_behavior,
                       SymbolT delimiter,
                       rmm::cuda_stream_view stream)
{
  check_input_size(json_in.size());

  // Range of encapsulating function that comprises:
  // -> DFA simulation for filtering out brackets and braces inside of quotes
  // -> Logical stack to infer the stack context
  CUDF_FUNC_RANGE();

  // Symbol representing the JSON-root (i.e., we're at nesting level '0')
  constexpr StackSymbolT root_symbol = '_';
  // This can be any stack symbol from the stack alphabet that does not push onto stack
  constexpr StackSymbolT read_symbol = 'x';

  // Number of stack operations in the input (i.e., number of '{', '}', '[', ']' outside of quotes)
  rmm::device_scalar<SymbolOffsetT> d_num_stack_ops(stream);

  // Sequence of stack symbols and their position in the original input (sparse representation)
  rmm::device_uvector<StackSymbolT> stack_ops{json_in.size(), stream};
  rmm::device_uvector<SymbolOffsetT> stack_op_indices{json_in.size(), stream};

  // Prepare finite-state transducer that only selects '{', '}', '[', ']' outside of quotes
  constexpr auto max_translation_table_size =
    to_stack_op::NUM_SYMBOL_GROUPS * to_stack_op::TT_NUM_STATES;

  auto json_to_stack_ops_fst = fst::detail::make_fst(
    fst::detail::make_symbol_group_lut(to_stack_op::get_sgid_lut(delimiter)),
    fst::detail::make_transition_table(to_stack_op::get_transition_table(stack_behavior)),
    fst::detail::make_translation_table<max_translation_table_size>(
      to_stack_op::get_translation_table(stack_behavior)),
    stream);

  // "Search" for relevant occurrence of brackets and braces that indicate the beginning/end
  // of structs/lists
  json_to_stack_ops_fst.Transduce(json_in.begin(),
                                  static_cast<SymbolOffsetT>(json_in.size()),
                                  stack_ops.data(),
                                  stack_op_indices.data(),
                                  d_num_stack_ops.data(),
                                  to_stack_op::start_state,
                                  stream);

  // Copy back to actual number of stack operations
  auto const num_stack_ops = d_num_stack_ops.value(stream);

  // Stack operations with indices are converted to top of the stack for each character in the input
  if (stack_behavior == stack_behavior_t::ResetOnDelimiter) {
    fst::sparse_stack_op_to_top_of_stack<fst::stack_op_support::WITH_RESET_SUPPORT, StackLevelT>(
      stack_ops.data(),
      device_span<SymbolOffsetT>{stack_op_indices.data(), num_stack_ops},
      JSONWithRecoveryToStackOp{},
      d_top_of_stack,
      root_symbol,
      read_symbol,
      json_in.size(),
      stream);
  } else {
    fst::sparse_stack_op_to_top_of_stack<fst::stack_op_support::NO_RESET_SUPPORT, StackLevelT>(
      stack_ops.data(),
      device_span<SymbolOffsetT>{stack_op_indices.data(), num_stack_ops},
      JSONToStackOp{},
      d_top_of_stack,
      root_symbol,
      read_symbol,
      json_in.size(),
      stream);
  }
}

std::pair<rmm::device_uvector<PdaTokenT>, rmm::device_uvector<SymbolOffsetT>> process_token_stream(
  device_span<PdaTokenT const> tokens,
  device_span<SymbolOffsetT const> token_indices,
  rmm::cuda_stream_view stream)
{
  // Instantiate FST for post-processing the token stream to remove all tokens that belong to an
  // invalid JSON line
  token_filter::UnwrapTokenFromSymbolOp sgid_op{};
  auto filter_fst =
    fst::detail::make_fst(fst::detail::make_symbol_group_lut(token_filter::symbol_groups, sgid_op),
                          fst::detail::make_transition_table(token_filter::transition_table),
                          fst::detail::make_translation_functor(token_filter::TransduceToken{}),
                          stream);

  auto const mr = rmm::mr::get_current_device_resource();
  rmm::device_scalar<SymbolOffsetT> d_num_selected_tokens(stream, mr);
  rmm::device_uvector<PdaTokenT> filtered_tokens_out{tokens.size(), stream, mr};
  rmm::device_uvector<SymbolOffsetT> filtered_token_indices_out{tokens.size(), stream, mr};

  // The FST is run on the reverse token stream, discarding all tokens between ErrorBegin and the
  // next LineEnd (LineEnd, inv_token_0, inv_token_1, ..., inv_token_n, ErrorBegin, LineEnd, ...),
  // emitting a [StructBegin, StructEnd] pair on the end of such an invalid line. In that example,
  // inv_token_i for i in [0, n] together with the ErrorBegin are removed and replaced with
  // StructBegin, StructEnd. Also, all LineEnd are removed as well, as these are not relevant after
  // this stage anymore
  filter_fst.Transduce(
    thrust::make_reverse_iterator(thrust::make_zip_iterator(tokens.data(), token_indices.data()) +
                                  tokens.size()),
    static_cast<SymbolOffsetT>(tokens.size()),
    thrust::make_reverse_iterator(
      thrust::make_zip_iterator(filtered_tokens_out.data(), filtered_token_indices_out.data()) +
      tokens.size()),
    thrust::make_discard_iterator(),
    d_num_selected_tokens.data(),
    token_filter::start_state,
    stream);

  auto const num_total_tokens = d_num_selected_tokens.value(stream);
  rmm::device_uvector<PdaTokenT> tokens_out{num_total_tokens, stream, mr};
  rmm::device_uvector<SymbolOffsetT> token_indices_out{num_total_tokens, stream, mr};
  thrust::copy(rmm::exec_policy(stream),
               filtered_tokens_out.end() - num_total_tokens,
               filtered_tokens_out.end(),
               tokens_out.data());
  thrust::copy(rmm::exec_policy(stream),
               filtered_token_indices_out.end() - num_total_tokens,
               filtered_token_indices_out.end(),
               token_indices_out.data());

  return std::make_pair(std::move(tokens_out), std::move(token_indices_out));
}

std::pair<rmm::device_uvector<PdaTokenT>, rmm::device_uvector<SymbolOffsetT>> get_token_stream(
  device_span<SymbolT const> json_in,
  cudf::io::json_reader_options const& options,
  rmm::cuda_stream_view stream,
  rmm::device_async_resource_ref mr)
{
  check_input_size(json_in.size());

  // Range of encapsulating function that parses to internal columnar data representation
  CUDF_FUNC_RANGE();

  auto const delimited_json = options.is_enabled_lines();
  auto const delimiter      = options.get_delimiter();

  // (!delimited_json)                         => JSON
  // (delimited_json and recover_from_error)   => JSON_LINES_RECOVER
  // (delimited_json and !recover_from_error)  => JSON_LINES
  auto format = delimited_json ? (options.recovery_mode() == json_recovery_mode_t::RECOVER_WITH_NULL
                                    ? tokenizer_pda::json_format_cfg_t::JSON_LINES_RECOVER
                                    : tokenizer_pda::json_format_cfg_t::JSON_LINES)
                               : tokenizer_pda::json_format_cfg_t::JSON;

  // Prepare for PDA transducer pass, merging input symbols with stack symbols
  auto const recover_from_error = (format == tokenizer_pda::json_format_cfg_t::JSON_LINES_RECOVER);

  // Memory holding the top-of-stack stack context for the input
  rmm::device_uvector<StackSymbolT> stack_symbols{json_in.size(), stream};

  // Identify what is the stack context for each input character (JSON-root, struct, or list)
  auto const stack_behavior =
    recover_from_error ? stack_behavior_t::ResetOnDelimiter : stack_behavior_t::PushPopWithoutReset;
  get_stack_context(json_in, stack_symbols.data(), stack_behavior, delimiter, stream);

  // Input to the full pushdown automaton finite-state transducer, where a input symbol comprises
  // the combination of a character from the JSON input together with the stack context for that
  // character.
  auto zip_in = thrust::make_zip_iterator(json_in.data(), stack_symbols.data());

  // Spark, as the main stakeholder in the `recover_from_error` option, has the specific need to
  // ignore any characters that follow the first value on each JSON line. This is an FST that
  // fixes the stack context for those excess characters. That is, that all those excess characters
  // will be interpreted in the root stack context
  if (recover_from_error) {
    auto fix_stack_of_excess_chars = fst::detail::make_fst(
      fst::detail::make_symbol_group_lookup_op(
        fix_stack_of_excess_chars::SymbolPairToSymbolGroupId{delimiter}),
      fst::detail::make_transition_table(fix_stack_of_excess_chars::transition_table),
      fst::detail::make_translation_functor(fix_stack_of_excess_chars::TransduceInputOp{}),
      stream);
    fix_stack_of_excess_chars.Transduce(zip_in,
                                        static_cast<SymbolOffsetT>(json_in.size()),
                                        stack_symbols.data(),
                                        thrust::make_discard_iterator(),
                                        thrust::make_discard_iterator(),
                                        fix_stack_of_excess_chars::start_state,
                                        stream);

    // Make sure memory of the FST's lookup tables isn't freed before the FST completes
    stream.synchronize();
  }

  constexpr auto max_translation_table_size =
    tokenizer_pda::NUM_PDA_SGIDS *
    static_cast<tokenizer_pda::StateT>(tokenizer_pda::pda_state_t::PD_NUM_STATES);

  auto json_to_tokens_fst = fst::detail::make_fst(
    fst::detail::make_symbol_group_lookup_op(tokenizer_pda::PdaSymbolToSymbolGroupId{delimiter}),
    fst::detail::make_transition_table(tokenizer_pda::get_transition_table(format)),
    fst::detail::make_translation_table<max_translation_table_size>(
      tokenizer_pda::get_translation_table(recover_from_error)),
    stream);

  // Perform a PDA-transducer pass
  // Compute the maximum amount of tokens that can possibly be emitted for a given input size
  // Worst case ratio of tokens per input char is given for a struct with an empty field name, that
  // may be arbitrarily deeply nested: {"":_}, where '_' is a placeholder for any JSON value,
  // possibly another such struct. That is, 6 tokens for 5 chars (plus chars and tokens of '_')
  std::size_t constexpr min_chars_per_struct  = 5;
  std::size_t constexpr max_tokens_per_struct = 6;
  auto const max_token_out_count =
    cudf::util::div_rounding_up_safe(json_in.size(), min_chars_per_struct) * max_tokens_per_struct;
  rmm::device_scalar<std::size_t> num_written_tokens{stream};
  // In case we're recovering on invalid JSON lines, post-processing the token stream requires to
  // see a JSON-line delimiter as the very first item
  SymbolOffsetT const delimiter_offset =
    (format == tokenizer_pda::json_format_cfg_t::JSON_LINES_RECOVER ? 1 : 0);
  rmm::device_uvector<PdaTokenT> tokens{max_token_out_count + delimiter_offset, stream, mr};
  rmm::device_uvector<SymbolOffsetT> tokens_indices{
    max_token_out_count + delimiter_offset, stream, mr};

  json_to_tokens_fst.Transduce(zip_in,
                               static_cast<SymbolOffsetT>(json_in.size()),
                               tokens.data() + delimiter_offset,
                               tokens_indices.data() + delimiter_offset,
                               num_written_tokens.data(),
                               tokenizer_pda::start_state,
                               stream);

  auto const num_total_tokens = num_written_tokens.value(stream) + delimiter_offset;
  tokens.resize(num_total_tokens, stream);
  tokens_indices.resize(num_total_tokens, stream);

  if (delimiter_offset == 1) {
    tokens.set_element(0, token_t::LineEnd, stream);
    auto [filtered_tokens, filtered_tokens_indices] =
      process_token_stream(tokens, tokens_indices, stream);
    tokens         = std::move(filtered_tokens);
    tokens_indices = std::move(filtered_tokens_indices);
  }

  CUDF_EXPECTS(num_total_tokens <= max_token_out_count,
               "Generated token count exceeds the expected token count");

  return std::make_pair(std::move(tokens), std::move(tokens_indices));
}

/**
 * @brief Parses the given JSON string and generates a tree representation of the given input.
 *
 * @param[in,out] root_column The root column of the hierarchy of columns into which data is parsed
 * @param[in,out] current_data_path The stack represents the path from the JSON root node to the
 * first node encountered in \p input
 * @param[in] input The JSON input in host memory
 * @param[in] d_input The JSON input in device memory
 * @param[in] options Parsing options specifying the parsing behaviour
 * @param[in] include_quote_char Whether to include the original quote chars around string values,
 * allowing to distinguish string values from numeric and literal values
 * @param[in] stream The CUDA stream to which kernels are dispatched
 * @param[in] mr Optional, resource with which to allocate
 * @return The columnar representation of the data from the given JSON input
 */
void make_json_column(json_column& root_column,
                      std::stack<tree_node>& current_data_path,
                      host_span<SymbolT const> input,
                      device_span<SymbolT const> d_input,
                      cudf::io::json_reader_options const& options,
                      bool include_quote_char,
                      rmm::cuda_stream_view stream,
                      rmm::device_async_resource_ref mr)
{
  // Range of encapsulating function that parses to internal columnar data representation
  CUDF_FUNC_RANGE();

  // Parse the JSON and get the token stream
  auto const [d_tokens_gpu, d_token_indices_gpu] = get_token_stream(d_input, options, stream, mr);

  // Copy the JSON tokens to the host
  thrust::host_vector<PdaTokenT> tokens =
    cudf::detail::make_host_vector_async(d_tokens_gpu, stream);
  thrust::host_vector<SymbolOffsetT> token_indices_gpu =
    cudf::detail::make_host_vector_async(d_token_indices_gpu, stream);

  // Make sure tokens have been copied to the host
  stream.synchronize();

  // Whether this token is the valid token to begin the JSON document with
  auto is_valid_root_token = [](PdaTokenT const token) {
    switch (token) {
      case token_t::StructBegin:
      case token_t::ListBegin:
      case token_t::StringBegin:
      case token_t::ValueBegin: return true;
      default: return false;
    };
  };

  // Returns the token's corresponding column type
  auto token_to_column_type = [](PdaTokenT const token) {
    switch (token) {
      case token_t::StructBegin: return json_col_t::StructColumn;
      case token_t::ListBegin: return json_col_t::ListColumn;
      case token_t::StringBegin: return json_col_t::StringColumn;
      case token_t::ValueBegin: return json_col_t::StringColumn;
      default: return json_col_t::Unknown;
    };
  };

  // Depending on whether we want to include the quotes of strings or not, respectively, we:
  // (a) strip off the beginning quote included in StringBegin and FieldNameBegin or
  // (b) include of the end quote excluded from in StringEnd and strip off the beginning quote
  // included FieldNameBegin
  auto get_token_index = [include_quote_char](PdaTokenT const token,
                                              SymbolOffsetT const token_index) {
    constexpr SymbolOffsetT quote_char_size = 1;
    switch (token) {
      // Optionally strip off quote char included for StringBegin
      case token_t::StringBegin: return token_index + (include_quote_char ? 0 : quote_char_size);
      // Optionally include trailing quote char for string values excluded for StringEnd
      case token_t::StringEnd: return token_index + (include_quote_char ? quote_char_size : 0);
      // Strip off quote char included for FieldNameBegin
      case token_t::FieldNameBegin: return token_index + quote_char_size;
      default: return token_index;
    };
  };

  // The end-of-* partner token for a given beginning-of-* token
  auto end_of_partner = [](PdaTokenT const token) {
    switch (token) {
      case token_t::StringBegin: return token_t::StringEnd;
      case token_t::ValueBegin: return token_t::ValueEnd;
      case token_t::FieldNameBegin: return token_t::FieldNameEnd;
      default: return token_t::ErrorBegin;
    };
  };

#ifdef NJP_DEBUG_PRINT
  auto column_type_string = [](json_col_t column_type) {
    switch (column_type) {
      case json_col_t::Unknown: return "Unknown";
      case json_col_t::ListColumn: return "List";
      case json_col_t::StructColumn: return "Struct";
      case json_col_t::StringColumn: return "String";
      default: return "Unknown";
    }
  };

  auto token_to_string = [](PdaTokenT token_type) {
    switch (token_type) {
      case token_t::StructBegin: return "StructBegin";
      case token_t::StructEnd: return "StructEnd";
      case token_t::ListBegin: return "ListBegin";
      case token_t::ListEnd: return "ListEnd";
      case token_t::StructMemberBegin: return "StructMemberBegin";
      case token_t::StructMemberEnd: return "StructMemberEnd";
      case token_t::FieldNameBegin: return "FieldNameBegin";
      case token_t::FieldNameEnd: return "FieldNameEnd";
      case token_t::StringBegin: return "StringBegin";
      case token_t::StringEnd: return "StringEnd";
      case token_t::ValueBegin: return "ValueBegin";
      case token_t::ValueEnd: return "ValueEnd";
      case token_t::ErrorBegin: return "ErrorBegin";
      case token_t::LineEnd: return "LineEnd";
      default: return "Unknown";
    }
  };
#endif

  /**
   * @brief Updates the given row in the given column with a new string_end and child_count. In
   * particular, updating the child count is relevant for list columns.
   */
  auto update_row =
    [](json_column* column, uint32_t row_index, uint32_t string_end, uint32_t child_count) {
#ifdef NJP_DEBUG_PRINT
      std::cout << "  -> update_row()\n";
      std::cout << "  ---> col@" << column << "\n";
      std::cout << "  ---> row #" << row_index << "\n";
      std::cout << "  ---> string_lengths = " << (string_end - column->string_offsets[row_index])
                << "\n";
      std::cout << "  ---> child_offsets = " << (column->child_offsets[row_index + 1] + child_count)
                << "\n";
#endif
      column->string_lengths[row_index]    = column->child_offsets[row_index + 1] + child_count;
      column->child_offsets[row_index + 1] = column->child_offsets[row_index + 1] + child_count;
    };

  /**
   * @brief Gets the currently selected child column given a \p current_data_path.
   *
   * That is, if \p current_data_path top-of-stack is
   * (a) a struct, the selected child column corresponds to the child column of the last field name
   * node encountered.
   * (b) a list, the selected child column corresponds to single child column of
   * the list column. In this case, the child column may not exist yet.
   */
  auto get_selected_column = [](std::stack<tree_node>& current_data_path) {
    json_column* selected_col = current_data_path.top().current_selected_col;

    // If the node does not have a selected column yet
    if (selected_col == nullptr) {
      // We're looking at the child column of a list column
      if (current_data_path.top().column->type == json_col_t::ListColumn) {
        CUDF_EXPECTS(current_data_path.top().column->child_columns.size() <= 1,
                     "Encountered a list column with more than a single child column");
        // The child column has yet to be created
        if (current_data_path.top().column->child_columns.empty()) {
          current_data_path.top().column->child_columns.emplace(std::string{list_child_name},
                                                                json_column{json_col_t::Unknown});
          current_data_path.top().column->column_order.push_back(list_child_name);
        }
        current_data_path.top().current_selected_col =
          &current_data_path.top().column->child_columns.begin()->second;
        selected_col = current_data_path.top().current_selected_col;
      } else {
        CUDF_FAIL("Trying to retrieve child column without encountering a field name.");
      }
    }
#ifdef NJP_DEBUG_PRINT
    std::cout << "  -> get_selected_column()\n";
    std::cout << "  ---> selected col@" << selected_col << "\n";
#endif
    return selected_col;
  };

  /**
   * @brief Returns a pointer to the child column with the given \p field_name within the current
   * struct column.
   */
  auto select_column = [](std::stack<tree_node>& current_data_path, std::string const& field_name) {
#ifdef NJP_DEBUG_PRINT
    std::cout << "  -> select_column(" << field_name << ")\n";
#endif
    // The field name's parent struct node
    auto& current_struct_node = current_data_path.top();

    // Verify that the field name node is actually a child of a struct
    CUDF_EXPECTS(current_data_path.top().column->type == json_col_t::StructColumn,
                 "Invalid JSON token sequence");

    json_column* struct_col  = current_struct_node.column;
    auto const& child_col_it = struct_col->child_columns.find(field_name);

    // The field name's column exists already, select that as the struct node's currently selected
    // child column
    if (child_col_it != struct_col->child_columns.end()) { return &child_col_it->second; }

    // The field name's column does not exist yet, so we have to append the child column to the
    // struct column
    struct_col->column_order.push_back(field_name);
    return &struct_col->child_columns.emplace(field_name, json_column{}).first->second;
  };

  /**
   * @brief Gets the row offset at which to insert. I.e., for a child column of a list column, we
   * just have to append the row to the end. Otherwise we have to propagate the row offset from the
   * parent struct column.
   */
  auto get_target_row_index = [](std::stack<tree_node> const& current_data_path,
                                 json_column* target_column) {
#ifdef NJP_DEBUG_PRINT
    std::cout << " -> target row: "
              << ((current_data_path.top().column->type == json_col_t::ListColumn)
                    ? target_column->current_offset
                    : current_data_path.top().row_index)
              << "\n";
#endif
    return (current_data_path.top().column->type == json_col_t::ListColumn)
             ? target_column->current_offset
             : current_data_path.top().row_index;
  };

  // The offset of the token currently being processed
  std::size_t offset = 0;

  // Giving names to magic constants
  constexpr uint32_t zero_child_count = 0;

  CUDF_EXPECTS(tokens.size() == token_indices_gpu.size(),
               "Unexpected mismatch in number of token types and token indices");
  CUDF_EXPECTS(tokens.size() > 0, "Empty JSON input not supported");

  // The JSON root may only be a struct, list, string, or value node
  CUDF_EXPECTS(is_valid_root_token(tokens[offset]), "Invalid beginning of JSON document");

  while (offset < tokens.size()) {
    // Verify there's at least the JSON root node left on the stack to which we can append data
    CUDF_EXPECTS(current_data_path.size() > 0, "Invalid JSON structure");

    // Verify that the current node in the tree (which becomes this nodes parent) can have children
    CUDF_EXPECTS(current_data_path.top().column->type == json_col_t::ListColumn or
                   current_data_path.top().column->type == json_col_t::StructColumn,
                 "Invalid JSON structure");

    // The token we're currently parsing
    auto const& token = tokens[offset];

#ifdef NJP_DEBUG_PRINT
    std::cout << "[" << token_to_string(token) << "]\n";
#endif

    // StructBegin token
    if (token == token_t::StructBegin) {
      // Get this node's column. That is, the parent node's selected column:
      // (a) if parent is a list, then this will (create and) return the list's only child column
      // (b) if parent is a struct, then this will return the column selected by the last field name
      // encountered.
      json_column* selected_col = get_selected_column(current_data_path);

      // Get the row offset at which to insert
      auto const target_row_index = get_target_row_index(current_data_path, selected_col);

      // Increment parent's child count and insert this struct node into the data path
      current_data_path.top().num_children++;
      current_data_path.push({selected_col, target_row_index, nullptr, zero_child_count});

      // Add this struct node to the current column
      selected_col->append_row(target_row_index,
                               token_to_column_type(tokens[offset]),
                               get_token_index(tokens[offset], token_indices_gpu[offset]),
                               get_token_index(tokens[offset], token_indices_gpu[offset]),
                               zero_child_count);
    }

    // StructEnd token
    else if (token == token_t::StructEnd) {
      // Verify that this node in fact a struct node (i.e., it was part of a struct column)
      CUDF_EXPECTS(current_data_path.top().column->type == json_col_t::StructColumn,
                   "Broken invariant while parsing JSON");
      CUDF_EXPECTS(current_data_path.top().column != nullptr,
                   "Broken invariant while parsing JSON");

      // Update row to account for string offset
      update_row(current_data_path.top().column,
                 current_data_path.top().row_index,
                 get_token_index(tokens[offset], token_indices_gpu[offset]),
                 current_data_path.top().num_children);

      // Pop struct from the path stack
      current_data_path.pop();
    }

    // ListBegin token
    else if (token == token_t::ListBegin) {
      // Get the selected column
      json_column* selected_col = get_selected_column(current_data_path);

      // Get the row offset at which to insert
      auto const target_row_index = get_target_row_index(current_data_path, selected_col);

      // Increment parent's child count and insert this struct node into the data path
      current_data_path.top().num_children++;
      current_data_path.push({selected_col, target_row_index, nullptr, zero_child_count});

      // Add this struct node to the current column
      selected_col->append_row(target_row_index,
                               token_to_column_type(tokens[offset]),
                               get_token_index(tokens[offset], token_indices_gpu[offset]),
                               get_token_index(tokens[offset], token_indices_gpu[offset]),
                               zero_child_count);
    }

    // ListEnd token
    else if (token == token_t::ListEnd) {
      // Verify that this node in fact a list node (i.e., it was part of a list column)
      CUDF_EXPECTS(current_data_path.top().column->type == json_col_t::ListColumn,
                   "Broken invariant while parsing JSON");
      CUDF_EXPECTS(current_data_path.top().column != nullptr,
                   "Broken invariant while parsing JSON");

      // Update row to account for string offset
      update_row(current_data_path.top().column,
                 current_data_path.top().row_index,
                 get_token_index(tokens[offset], token_indices_gpu[offset]),
                 current_data_path.top().num_children);

      // Pop list from the path stack
      current_data_path.pop();
    }

    // Error token
    else if (token == token_t::ErrorBegin) {
#ifdef NJP_DEBUG_PRINT
      std::cout << "[ErrorBegin]\n";
      std::cout << "@" << get_token_index(tokens[offset], token_indices_gpu[offset]);
#endif
      CUDF_FAIL("Parser encountered an invalid format.");
    }

    // FieldName, String, or Value (begin, end)-pair
    else if (token == token_t::FieldNameBegin or token == token_t::StringBegin or
             token == token_t::ValueBegin) {
      // Verify that this token has the right successor to build a correct (being, end) token pair
      CUDF_EXPECTS((offset + 1) < tokens.size(), "Invalid JSON token sequence");
      CUDF_EXPECTS(tokens[offset + 1] == end_of_partner(token), "Invalid JSON token sequence");

      // The offset to the first symbol from the JSON input associated with the current token
      auto const& token_begin_offset = get_token_index(tokens[offset], token_indices_gpu[offset]);

      // The offset to one past the last symbol associated with the current token
      auto const& token_end_offset =
        get_token_index(tokens[offset + 1], token_indices_gpu[offset + 1]);

      // FieldNameBegin
      // For the current struct node in the tree, select the child column corresponding to this
      // field name
      if (token == token_t::FieldNameBegin) {
        std::string field_name{input.data() + token_begin_offset,
                               (token_end_offset - token_begin_offset)};
        current_data_path.top().current_selected_col = select_column(current_data_path, field_name);
      }
      // StringBegin
      // ValueBegin
      // As we currently parse to string columns there's no further differentiation
      else if (token == token_t::StringBegin or token == token_t::ValueBegin) {
        // Get the selected column
        json_column* selected_col = get_selected_column(current_data_path);

        // Get the row offset at which to insert
        auto const target_row_index = get_target_row_index(current_data_path, selected_col);

        current_data_path.top().num_children++;

        selected_col->append_row(target_row_index,
                                 token_to_column_type(token),
                                 token_begin_offset,
                                 token_end_offset,
                                 zero_child_count);
      } else {
        CUDF_FAIL("Unknown JSON token");
      }

      // As we've also consumed the end-of-* token, we advance the processed token offset by one
      offset++;
    }

    offset++;
  }

  // Make sure all of a struct's child columns have the same length
  root_column.level_child_cols_recursively(root_column.current_offset);
}

/**
 * @brief Retrieves the parse_options to be used for type inference and type casting
 *
 * @param options The reader options to influence the relevant type inference and type casting
 * options
 * @param stream The CUDA stream to which kernels are dispatched
 */
cudf::io::parse_options parsing_options(cudf::io::json_reader_options const& options,
                                        rmm::cuda_stream_view stream)
{
  auto parse_opts = cudf::io::parse_options{',', '\n', '\"', '.'};

  parse_opts.dayfirst   = options.is_enabled_dayfirst();
  parse_opts.keepquotes = options.is_enabled_keep_quotes();
  parse_opts.trie_true  = cudf::detail::create_serialized_trie({"true"}, stream);
  parse_opts.trie_false = cudf::detail::create_serialized_trie({"false"}, stream);
  parse_opts.trie_na    = cudf::detail::create_serialized_trie({"", "null"}, stream);
  return parse_opts;
}

std::pair<std::unique_ptr<column>, std::vector<column_name_info>> json_column_to_cudf_column(
  json_column const& json_col,
  device_span<SymbolT const> d_input,
  cudf::io::json_reader_options const& options,
  std::optional<schema_element> schema,
  rmm::cuda_stream_view stream,
  rmm::device_async_resource_ref mr)
{
  // Range of orchestrating/encapsulating function
  CUDF_FUNC_RANGE();

  auto make_validity =
    [stream, mr](json_column const& json_col) -> std::pair<rmm::device_buffer, size_type> {
    auto const null_count = json_col.current_offset - json_col.valid_count;
    if (null_count == 0) { return {rmm::device_buffer{}, null_count}; }
    return {rmm::device_buffer{json_col.validity.data(),
                               bitmask_allocation_size_bytes(json_col.current_offset),
                               stream,
                               mr},
            null_count};
  };

  auto get_child_schema = [schema](auto child_name) -> std::optional<schema_element> {
    if (schema.has_value()) {
      auto const result = schema.value().child_types.find(child_name);
      if (result != std::end(schema.value().child_types)) { return result->second; }
    }
    return {};
  };

  switch (json_col.type) {
    case json_col_t::StringColumn: {
      auto const col_size = json_col.string_offsets.size();
      CUDF_EXPECTS(json_col.string_offsets.size() == json_col.string_lengths.size(),
                   "string offset, string length mismatch");

      // Move string_offsets and string_lengths to GPU
      rmm::device_uvector<json_column::row_offset_t> d_string_offsets =
        cudf::detail::make_device_uvector_async(
          json_col.string_offsets, stream, rmm::mr::get_current_device_resource());
      rmm::device_uvector<json_column::row_offset_t> d_string_lengths =
        cudf::detail::make_device_uvector_async(
          json_col.string_lengths, stream, rmm::mr::get_current_device_resource());

      // Prepare iterator that returns (string_offset, string_length)-tuples
      auto offset_length_it =
        thrust::make_zip_iterator(d_string_offsets.begin(), d_string_lengths.begin());

      data_type target_type{};

      if (schema.has_value()) {
#ifdef NJP_DEBUG_PRINT
        std::cout << "-> explicit type: "
                  << (schema.has_value() ? std::to_string(static_cast<int>(schema->type.id()))
                                         : "n/a");
#endif
        target_type = schema.value().type;
      }
      // Infer column type, if we don't have an explicit type for it
      else {
        target_type =
          cudf::io::detail::infer_data_type(parsing_options(options, stream).json_view(),
                                            d_input,
                                            offset_length_it,
                                            col_size,
                                            stream);
      }

      auto [result_bitmask, null_count] = make_validity(json_col);

      // Convert strings to the inferred data type
      auto col = parse_data(d_input.data(),
                            offset_length_it,
                            col_size,
                            target_type,
                            std::move(result_bitmask),
                            null_count,
                            parsing_options(options, stream).view(),
                            stream,
                            mr);

      // Reset nullable if we do not have nulls
      // This is to match the existing JSON reader's behaviour:
      // - Non-string columns will always be returned as nullable
      // - String columns will be returned as nullable, iff there's at least one null entry
      if (col->null_count() == 0) { col->set_null_mask(rmm::device_buffer{0, stream, mr}, 0); }

      // For string columns return ["offsets", "char"] schema
      if (target_type.id() == type_id::STRING) {
        return {std::move(col), std::vector<column_name_info>{{"offsets"}, {"chars"}}};
      }
      // Non-string leaf-columns (e.g., numeric) do not have child columns in the schema
      else {
        return {std::move(col), std::vector<column_name_info>{}};
      }
      break;
    }
    case json_col_t::StructColumn: {
      std::vector<std::unique_ptr<column>> child_columns;
      std::vector<column_name_info> column_names{};
      size_type num_rows{json_col.current_offset};
      // Create children columns
      for (auto const& col_name : json_col.column_order) {
        auto const& col = json_col.child_columns.find(col_name);
        column_names.emplace_back(col->first);
        auto const& child_col      = col->second;
        auto [child_column, names] = json_column_to_cudf_column(
          child_col, d_input, options, get_child_schema(col_name), stream, mr);
        CUDF_EXPECTS(num_rows == child_column->size(),
                     "All children columns must have the same size");
        child_columns.push_back(std::move(child_column));
        column_names.back().children = names;
      }
      auto [result_bitmask, null_count] = make_validity(json_col);
      return {
        make_structs_column(
          num_rows, std::move(child_columns), null_count, std::move(result_bitmask), stream, mr),
        column_names};
      break;
    }
    case json_col_t::ListColumn: {
      size_type num_rows = json_col.child_offsets.size();
      std::vector<column_name_info> column_names{};
      column_names.emplace_back("offsets");
      column_names.emplace_back(
        json_col.child_columns.empty() ? list_child_name : json_col.child_columns.begin()->first);

      rmm::device_uvector<json_column::row_offset_t> d_offsets =
        cudf::detail::make_device_uvector_async(json_col.child_offsets, stream, mr);
      auto offsets_column = std::make_unique<column>(
        data_type{type_id::INT32}, num_rows, d_offsets.release(), rmm::device_buffer{}, 0);
      // Create children column
      auto [child_column, names] =
        json_col.child_columns.empty()
          ? std::pair<std::unique_ptr<column>,
                      std::vector<column_name_info>>{std::make_unique<column>(),
                                                     std::vector<column_name_info>{}}
          : json_column_to_cudf_column(json_col.child_columns.begin()->second,
                                       d_input,
                                       options,
                                       get_child_schema(json_col.child_columns.begin()->first),
                                       stream,
                                       mr);
      column_names.back().children      = names;
      auto [result_bitmask, null_count] = make_validity(json_col);
      return {make_lists_column(num_rows - 1,
                                std::move(offsets_column),
                                std::move(child_column),
                                null_count,
                                std::move(result_bitmask),
                                stream,
                                mr),
              std::move(column_names)};
      break;
    }
    default: CUDF_FAIL("Unsupported column type, yet to be implemented"); break;
  }

  return {};
}

table_with_metadata host_parse_nested_json(device_span<SymbolT const> d_input,
                                           cudf::io::json_reader_options const& options,
                                           rmm::cuda_stream_view stream,
                                           rmm::device_async_resource_ref mr)
{
  // Range of orchestrating/encapsulating function
  CUDF_FUNC_RANGE();

  auto const h_input = cudf::detail::make_std_vector_async(d_input, stream);

  auto const new_line_delimited_json = options.is_enabled_lines();

  // Get internal JSON column
  json_column root_column{};
  std::stack<tree_node> data_path{};

  constexpr uint32_t row_offset_zero            = 0;
  constexpr uint32_t token_begin_offset_zero    = 0;
  constexpr uint32_t token_end_offset_zero      = 0;
  constexpr uint32_t node_init_child_count_zero = 0;

  // Whether the tokenizer stage should keep quote characters for string values
  // If the tokenizer keeps the quote characters, they may be stripped during type casting
  constexpr bool include_quote_chars = true;

  // We initialize the very root node and root column, which represent the JSON document being
  // parsed. That root node is a list node and that root column is a list column. The column has the
  // root node as its only row. The values parsed from the JSON input will be treated as follows:
  // (1) For JSON lines: we expect to find a list of JSON values that all
  // will be inserted into this root list column. (2) For regular JSON: we expect to have only a
  // single value (list, struct, string, number, literal) that will be inserted into this root
  // column.
  root_column.append_row(
    row_offset_zero, json_col_t::ListColumn, token_begin_offset_zero, token_end_offset_zero, 1);

  // Push the root node onto the stack for the data path
  data_path.push({&root_column, row_offset_zero, nullptr, node_init_child_count_zero});

  make_json_column(
    root_column, data_path, h_input, d_input, options, include_quote_chars, stream, mr);

  // data_root refers to the root column of the data represented by the given JSON string
  auto const& data_root =
    new_line_delimited_json ? root_column : root_column.child_columns.begin()->second;

  // Zero row entries
  if (data_root.type == json_col_t::ListColumn && data_root.child_columns.empty()) {
    return table_with_metadata{std::make_unique<table>(std::vector<std::unique_ptr<column>>{})};
  }

  // Verify that we were in fact given a list of structs (or in JSON speech: an array of objects)
  auto constexpr single_child_col_count = 1;
  CUDF_EXPECTS(data_root.type == json_col_t::ListColumn and
                 data_root.child_columns.size() == single_child_col_count and
                 data_root.child_columns.begin()->second.type == json_col_t::StructColumn,
               "Currently the nested JSON parser only supports an array of (nested) objects");

  // Slice off the root list column, which has only a single row that contains all the structs
  auto const& root_struct_col = data_root.child_columns.begin()->second;

  // Initialize meta data to be populated while recursing through the tree of columns
  std::vector<std::unique_ptr<column>> out_columns;
  std::vector<column_name_info> out_column_names;

  // Iterate over the struct's child columns and convert to cudf column
  size_type column_index = 0;
  for (auto const& col_name : root_struct_col.column_order) {
    auto const& json_col = root_struct_col.child_columns.find(col_name)->second;
    // Insert this columns name into the schema
    out_column_names.emplace_back(col_name);

    std::optional<schema_element> child_schema_element = std::visit(
      cudf::detail::visitor_overload{
        [column_index](std::vector<data_type> const& user_dtypes) -> std::optional<schema_element> {
          auto ret = (static_cast<std::size_t>(column_index) < user_dtypes.size())
                       ? std::optional<schema_element>{{user_dtypes[column_index]}}
                       : std::optional<schema_element>{};
#ifdef NJP_DEBUG_PRINT
          std::cout << "Column by index: #" << column_index << ", type id: "
                    << (ret.has_value() ? std::to_string(static_cast<int>(ret->type.id())) : "n/a")
                    << ", with " << (ret.has_value() ? ret->child_types.size() : 0) << " children"
                    << "\n";
#endif
          return ret;
        },
        [col_name](
          std::map<std::string, data_type> const& user_dtypes) -> std::optional<schema_element> {
          auto ret = (user_dtypes.find(col_name) != std::end(user_dtypes))
                       ? std::optional<schema_element>{{user_dtypes.find(col_name)->second}}
                       : std::optional<schema_element>{};
#ifdef NJP_DEBUG_PRINT
          std::cout << "Column by flat name: '" << col_name << "', type id: "
                    << (ret.has_value() ? std::to_string(static_cast<int>(ret->type.id())) : "n/a")
                    << ", with " << (ret.has_value() ? ret->child_types.size() : 0) << " children"
                    << "\n";
#endif
          return ret;
        },
        [col_name](std::map<std::string, schema_element> const& user_dtypes)
          -> std::optional<schema_element> {
          auto ret = (user_dtypes.find(col_name) != std::end(user_dtypes))
                       ? user_dtypes.find(col_name)->second
                       : std::optional<schema_element>{};
#ifdef NJP_DEBUG_PRINT
          std::cout << "Column by nested name: #" << col_name << ", type id: "
                    << (ret.has_value() ? std::to_string(static_cast<int>(ret->type.id())) : "n/a")
                    << ", with " << (ret.has_value() ? ret->child_types.size() : 0) << " children"
                    << "\n";
#endif
          return ret;
        }},
      options.get_dtypes());

    // Get this JSON column's cudf column and schema info
    auto [cudf_col, col_name_info] =
      json_column_to_cudf_column(json_col, d_input, options, child_schema_element, stream, mr);
    out_column_names.back().children = std::move(col_name_info);
    out_columns.emplace_back(std::move(cudf_col));

    column_index++;
  }

  return table_with_metadata{std::make_unique<table>(std::move(out_columns)), {out_column_names}};
}

}  // namespace detail
}  // namespace cudf::io::json

// Debug print flag
#undef NJP_DEBUG_PRINT
