#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2022, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "nested_json.hpp"

#include <io/fst/logical_stack.cuh>
#include <io/fst/lookup_tables.cuh>
#include <io/utilities/hostdevice_vector.hpp>

#include <cudf/column/column_factories.hpp>
#include <cudf/detail/utilities/vector_factories.hpp>
#include <cudf/detail/valid_if.cuh>
#include <cudf/table/table.hpp>
#include <cudf/types.hpp>
#include <cudf/utilities/bit.hpp>
#include <cudf/utilities/error.hpp>
#include <cudf/utilities/span.hpp>

#include <rmm/device_uvector.hpp>
#include <rmm/exec_policy.hpp>

#include <thrust/device_vector.h>
#include <thrust/transform.h>

#include <stack>

// Debug print flag
#ifndef NJP_DEBUG_PRINT
//#define NJP_DEBUG_PRINT
#endif

namespace {

/**
 * @brief While parsing the token stream, we use a stack of tree_nodes to maintain all the
 * information about the data path that is relevant.
 */
struct tree_node {
  // The column that this node is associated with
  cudf::io::json::json_column* column;

  // The row offset that this node belongs to within the given column
  uint32_t row_index;

  // Selected child column
  // E.g., if this is a struct node, and we subsequently encountered the field name "a", then this
  // point's to the struct's "a" child column
  cudf::io::json::json_column* current_selected_col = nullptr;

  std::size_t num_children = 0;
};
}  // namespace

namespace cudf::io::json {

// JSON to stack operator DFA (Deterministic Finite Automata)
namespace to_stack_op {

// Type used to represent the target state in the transition table
using StateT = char;

/**
 * @brief Definition of the DFA's states
 */
enum class dfa_states : StateT {
  // The active state while outside of a string. When encountering an opening bracket or curly
  // brace, we push it onto the stack. When encountering a closing bracket or brace, we pop from the
  // stack.
  TT_OOS = 0U,

  // The active state while within a string (e.g., field name or a string value). We do not push or
  // pop from the stack while in this state.
  TT_STR,

  // The active state after encountering an escape symbol (e.g., '\'), while in the TT_STR state.
  TT_ESC,

  // Total number of states
  TT_NUM_STATES
};

// Aliases for readability of the transition table
constexpr auto TT_OOS = dfa_states::TT_OOS;
constexpr auto TT_STR = dfa_states::TT_STR;
constexpr auto TT_ESC = dfa_states::TT_ESC;

/**
 * @brief Definition of the symbol groups
 */
enum class dfa_symbol_group_id : uint8_t {
  OPENING_BRACE,     ///< Opening brace SG: {
  OPENING_BRACKET,   ///< Opening bracket SG: [
  CLOSING_BRACE,     ///< Closing brace SG: }
  CLOSING_BRACKET,   ///< Closing bracket SG: ]
  QUOTE_CHAR,        ///< Quote character SG: "
  ESCAPE_CHAR,       ///< Escape character SG: '\'
  OTHER_SYMBOLS,     ///< SG implicitly matching all other characters
  NUM_SYMBOL_GROUPS  ///< Total number of symbol groups
};

constexpr auto TT_NUM_STATES     = static_cast<StateT>(dfa_states::TT_NUM_STATES);
constexpr auto NUM_SYMBOL_GROUPS = static_cast<uint32_t>(dfa_symbol_group_id::NUM_SYMBOL_GROUPS);

// The i-th string representing all the characters of a symbol group
std::array<std::string, NUM_SYMBOL_GROUPS - 1> const symbol_groups{
  {{"{"}, {"["}, {"}"}, {"]"}, {"\""}, {"\\"}}};

// Transition table
std::array<std::array<dfa_states, NUM_SYMBOL_GROUPS>, TT_NUM_STATES> const transition_table{
  {/* IN_STATE          {       [       }       ]       "       \    OTHER */
   /* TT_OOS    */ {{TT_OOS, TT_OOS, TT_OOS, TT_OOS, TT_STR, TT_OOS, TT_OOS}},
   /* TT_STR    */ {{TT_STR, TT_STR, TT_STR, TT_STR, TT_OOS, TT_ESC, TT_STR}},
   /* TT_ESC    */ {{TT_STR, TT_STR, TT_STR, TT_STR, TT_STR, TT_STR, TT_STR}}}};

// Translation table (i.e., for each transition, what are the symbols that we output)
std::array<std::array<std::vector<char>, NUM_SYMBOL_GROUPS>, TT_NUM_STATES> const translation_table{
  {/* IN_STATE         {      [      }      ]      "      \    OTHER */
   /* TT_OOS    */ {{{'{'}, {'['}, {'}'}, {']'}, {'x'}, {'x'}, {'x'}}},
   /* TT_STR    */ {{{'x'}, {'x'}, {'x'}, {'x'}, {'x'}, {'x'}, {'x'}}},
   /* TT_ESC    */ {{{'x'}, {'x'}, {'x'}, {'x'}, {'x'}, {'x'}, {'x'}}}}};

// The DFA's starting state
constexpr auto start_state = static_cast<StateT>(TT_OOS);
}  // namespace to_stack_op

// JSON tokenizer pushdown automaton
namespace tokenizer_pda {

// Type used to represent the target state in the transition table
using StateT = char;

/**
 * @brief Symbol groups for the input alphabet for the pushdown automaton
 */
enum class symbol_group_id : PdaSymbolGroupIdT {
  /// Opening brace
  OPENING_BRACE,
  /// Opening bracket
  OPENING_BRACKET,
  /// Closing brace
  CLOSING_BRACE,
  /// Closing bracket
  CLOSING_BRACKET,
  /// Quote
  QUOTE,
  /// Escape
  ESCAPE,
  /// Comma
  COMMA,
  /// Colon
  COLON,
  /// Whitespace
  WHITE_SPACE,
  /// Other (any input symbol not assigned to one of the above symbol groups)
  OTHER,
  /// Total number of symbol groups amongst which to differentiate
  NUM_PDA_INPUT_SGS
};

/**
 * @brief Symbols in the stack alphabet
 */
enum class stack_symbol_group_id : PdaStackSymbolGroupIdT {
  /// Symbol representing that we're at the JSON root (nesting level 0)
  STACK_ROOT,

  /// Symbol representing that we're currently within a list object
  STACK_LIST,

  /// Symbol representing that we're currently within a struct object
  STACK_STRUCT,

  /// Total number of symbols in the stack alphabet
  NUM_STACK_SGS
};
constexpr auto NUM_PDA_INPUT_SGS =
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::NUM_PDA_INPUT_SGS);
constexpr auto NUM_STACK_SGS =
  static_cast<PdaStackSymbolGroupIdT>(stack_symbol_group_id::NUM_STACK_SGS);

/// Total number of symbol groups to differentiate amongst (stack alphabet * input alphabet)
constexpr PdaSymbolGroupIdT NUM_PDA_SGIDS = NUM_PDA_INPUT_SGS * NUM_STACK_SGS;

/// Mapping a input symbol to the symbol group id
static __constant__ PdaSymbolGroupIdT tos_sg_to_pda_sgid[] = {
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::OTHER),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::OTHER),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::OTHER),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::OTHER),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::OTHER),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::OTHER),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::OTHER),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::OTHER),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::OTHER),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::WHITE_SPACE),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::WHITE_SPACE),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::OTHER),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::OTHER),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::WHITE_SPACE),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::OTHER),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::OTHER),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::OTHER),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::OTHER),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::OTHER),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::OTHER),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::OTHER),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::OTHER),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::OTHER),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::OTHER),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::OTHER),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::OTHER),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::OTHER),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::OTHER),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::OTHER),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::OTHER),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::OTHER),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::OTHER),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::WHITE_SPACE),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::OTHER),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::QUOTE),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::OTHER),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::OTHER),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::OTHER),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::OTHER),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::OTHER),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::OTHER),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::OTHER),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::OTHER),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::OTHER),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::COMMA),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::OTHER),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::OTHER),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::OTHER),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::OTHER),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::OTHER),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::OTHER),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::OTHER),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::OTHER),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::OTHER),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::OTHER),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::OTHER),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::OTHER),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::OTHER),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::COLON),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::OTHER),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::OTHER),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::OTHER),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::OTHER),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::OTHER),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::OTHER),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::OTHER),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::OTHER),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::OTHER),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::OTHER),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::OTHER),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::OTHER),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::OTHER),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::OTHER),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::OTHER),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::OTHER),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::OTHER),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::OTHER),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::OTHER),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::OTHER),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::OTHER),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::OTHER),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::OTHER),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::OTHER),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::OTHER),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::OTHER),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::OTHER),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::OTHER),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::OTHER),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::OTHER),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::OTHER),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::OTHER),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::OPENING_BRACKET),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::ESCAPE),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::CLOSING_BRACKET),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::OTHER),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::OTHER),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::OTHER),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::OTHER),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::OTHER),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::OTHER),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::OTHER),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::OTHER),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::OTHER),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::OTHER),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::OTHER),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::OTHER),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::OTHER),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::OTHER),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::OTHER),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::OTHER),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::OTHER),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::OTHER),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::OTHER),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::OTHER),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::OTHER),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::OTHER),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::OTHER),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::OTHER),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::OTHER),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::OTHER),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::OTHER),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::OTHER),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::OTHER),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::OPENING_BRACE),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::OTHER),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::CLOSING_BRACE),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::OTHER)};

/**
 * @brief Maps a (top-of-stack symbol, input symbol)-pair to a symbol group id of the deterministic
 * visibly pushdown automaton (DVPA)
 */
struct PdaSymbolToSymbolGroupId {
  template <typename SymbolT, typename StackSymbolT>
  __device__ __forceinline__ PdaSymbolGroupIdT
  operator()(thrust::tuple<SymbolT, StackSymbolT> symbol_pair)
  {
    // The symbol read from the input
    auto symbol = thrust::get<0>(symbol_pair);

    // The stack symbol (i.e., what is on top of the stack at the time the input symbol was read)
    // I.e., whether we're reading in something within a struct, a list, or the JSON root
    auto stack_symbol = thrust::get<1>(symbol_pair);

    // The stack symbol offset: '_' is the root group (0), '[' is the list group (1), '{' is the
    // struct group (2)
    int32_t stack_idx = static_cast<PdaStackSymbolGroupIdT>(
      (stack_symbol == '_') ? stack_symbol_group_id::STACK_ROOT
                            : ((stack_symbol == '[') ? stack_symbol_group_id::STACK_LIST
                                                     : stack_symbol_group_id::STACK_STRUCT));

    // The relative symbol group id of the current input symbol
    constexpr auto pda_sgid_lookup_size =
      static_cast<int32_t>(sizeof(tos_sg_to_pda_sgid) / sizeof(tos_sg_to_pda_sgid[0]));
    PdaSymbolGroupIdT symbol_gid =
      tos_sg_to_pda_sgid[min(static_cast<int32_t>(symbol), pda_sgid_lookup_size - 1)];
    return stack_idx * static_cast<PdaSymbolGroupIdT>(symbol_group_id::NUM_PDA_INPUT_SGS) +
           symbol_gid;
  }
};

// The states defined by the pushdown automaton
enum class pda_state_t : StateT {
  // Beginning of value
  PD_BOV,
  // Beginning of array
  PD_BOA,
  // Literal or number
  PD_LON,
  // String
  PD_STR,
  // After escape char when within string
  PD_SCE,
  // After having parsed a value
  PD_PVL,
  // Before the next field name
  PD_BFN,
  // Field name
  PD_FLN,
  // After escape char when within field name
  PD_FNE,
  // After a field name inside a struct
  PD_PFN,
  // Error state (trap state)
  PD_ERR,
  // Total number of PDA states
  PD_NUM_STATES
};

// Aliases for readability of the transition table
constexpr auto PD_BOV = pda_state_t::PD_BOV;
constexpr auto PD_BOA = pda_state_t::PD_BOA;
constexpr auto PD_LON = pda_state_t::PD_LON;
constexpr auto PD_STR = pda_state_t::PD_STR;
constexpr auto PD_SCE = pda_state_t::PD_SCE;
constexpr auto PD_PVL = pda_state_t::PD_PVL;
constexpr auto PD_BFN = pda_state_t::PD_BFN;
constexpr auto PD_FLN = pda_state_t::PD_FLN;
constexpr auto PD_FNE = pda_state_t::PD_FNE;
constexpr auto PD_PFN = pda_state_t::PD_PFN;
constexpr auto PD_ERR = pda_state_t::PD_ERR;

constexpr auto PD_NUM_STATES = static_cast<StateT>(pda_state_t::PD_NUM_STATES);

// The starting state of the pushdown automaton
constexpr auto start_state = static_cast<StateT>(pda_state_t::PD_BOV);

// Identity symbol to symbol group lookup table
std::vector<std::vector<char>> const pda_sgids{
  {0},  {1},  {2},  {3},  {4},  {5},  {6},  {7},  {8},  {9},  {10}, {11}, {12}, {13}, {14},
  {15}, {16}, {17}, {18}, {19}, {20}, {21}, {22}, {23}, {24}, {25}, {26}, {27}, {28}, {29}};

/**
 * @brief Getting the transition table
 */
auto get_transition_table()
{
  std::array<std::array<pda_state_t, NUM_PDA_SGIDS>, PD_NUM_STATES> pda_tt;
  //  {       [       }       ]       "       \       ,       :     space   other
  pda_tt[static_cast<StateT>(pda_state_t::PD_BOV)] = {
    PD_BOA, PD_BOA, PD_ERR, PD_ERR, PD_STR, PD_ERR, PD_ERR, PD_ERR, PD_BOV, PD_LON,
    PD_BOA, PD_BOA, PD_ERR, PD_ERR, PD_STR, PD_ERR, PD_ERR, PD_ERR, PD_BOV, PD_LON,
    PD_BOA, PD_BOA, PD_ERR, PD_ERR, PD_STR, PD_ERR, PD_ERR, PD_ERR, PD_BOV, PD_LON};
  pda_tt[static_cast<StateT>(pda_state_t::PD_BOA)] = {
    PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_ERR,
    PD_BOA, PD_BOA, PD_ERR, PD_PVL, PD_STR, PD_ERR, PD_ERR, PD_ERR, PD_BOA, PD_LON,
    PD_ERR, PD_ERR, PD_PVL, PD_ERR, PD_FLN, PD_ERR, PD_ERR, PD_ERR, PD_BOA, PD_ERR};
  pda_tt[static_cast<StateT>(pda_state_t::PD_LON)] = {
    PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_PVL, PD_LON,
    PD_ERR, PD_ERR, PD_ERR, PD_PVL, PD_ERR, PD_ERR, PD_BOV, PD_ERR, PD_PVL, PD_LON,
    PD_ERR, PD_ERR, PD_PVL, PD_ERR, PD_ERR, PD_ERR, PD_BFN, PD_ERR, PD_PVL, PD_LON};
  pda_tt[static_cast<StateT>(pda_state_t::PD_STR)] = {
    PD_STR, PD_STR, PD_STR, PD_STR, PD_PVL, PD_SCE, PD_STR, PD_STR, PD_STR, PD_STR,
    PD_STR, PD_STR, PD_STR, PD_STR, PD_PVL, PD_SCE, PD_STR, PD_STR, PD_STR, PD_STR,
    PD_STR, PD_STR, PD_STR, PD_STR, PD_PVL, PD_SCE, PD_STR, PD_STR, PD_STR, PD_STR};
  pda_tt[static_cast<StateT>(pda_state_t::PD_SCE)] = {
    PD_STR, PD_STR, PD_STR, PD_STR, PD_STR, PD_STR, PD_STR, PD_STR, PD_STR, PD_STR,
    PD_STR, PD_STR, PD_STR, PD_STR, PD_STR, PD_STR, PD_STR, PD_STR, PD_STR, PD_STR,
    PD_STR, PD_STR, PD_STR, PD_STR, PD_STR, PD_STR, PD_STR, PD_STR, PD_STR, PD_STR};
  pda_tt[static_cast<StateT>(pda_state_t::PD_PVL)] = {
    PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_PVL, PD_ERR,
    PD_ERR, PD_ERR, PD_ERR, PD_PVL, PD_ERR, PD_ERR, PD_BOV, PD_ERR, PD_PVL, PD_ERR,
    PD_ERR, PD_ERR, PD_PVL, PD_ERR, PD_ERR, PD_ERR, PD_BFN, PD_ERR, PD_PVL, PD_ERR};
  pda_tt[static_cast<StateT>(pda_state_t::PD_BFN)] = {
    PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_ERR,
    PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_ERR,
    PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_FLN, PD_ERR, PD_ERR, PD_ERR, PD_BFN, PD_ERR};
  pda_tt[static_cast<StateT>(pda_state_t::PD_FLN)] = {
    PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_ERR,
    PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_ERR,
    PD_FLN, PD_FLN, PD_FLN, PD_FLN, PD_PFN, PD_FNE, PD_FLN, PD_FLN, PD_FLN, PD_FLN};
  pda_tt[static_cast<StateT>(pda_state_t::PD_FNE)] = {
    PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_ERR,
    PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_ERR,
    PD_FLN, PD_FLN, PD_FLN, PD_FLN, PD_FLN, PD_FLN, PD_FLN, PD_FLN, PD_FLN, PD_FLN};
  pda_tt[static_cast<StateT>(pda_state_t::PD_PFN)] = {
    PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_ERR,
    PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_ERR,
    PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_BOV, PD_PFN, PD_ERR};
  pda_tt[static_cast<StateT>(pda_state_t::PD_ERR)] = {
    PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_ERR,
    PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_ERR,
    PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_ERR};
  return pda_tt;
}

/**
 * @brief Getting the translation table
 */
auto get_translation_table()
{
  std::array<std::array<std::vector<char>, NUM_PDA_SGIDS>, PD_NUM_STATES> pda_tlt;
  pda_tlt[static_cast<StateT>(pda_state_t::PD_BOV)] = {{{token_t::StructBegin},
                                                        {token_t::ListBegin},
                                                        {token_t::ErrorBegin},
                                                        {token_t::ErrorBegin},
                                                        {token_t::StringBegin},
                                                        {token_t::ErrorBegin},
                                                        {token_t::ErrorBegin},
                                                        {token_t::ErrorBegin},
                                                        {},
                                                        {token_t::ValueBegin},
                                                        {token_t::StructBegin},
                                                        {token_t::ListBegin},
                                                        {token_t::ErrorBegin},
                                                        {token_t::ErrorBegin},
                                                        {token_t::StringBegin},
                                                        {token_t::ErrorBegin},
                                                        {token_t::ErrorBegin},
                                                        {token_t::ErrorBegin},
                                                        {},
                                                        {token_t::ValueBegin},
                                                        {token_t::StructBegin},
                                                        {token_t::ListBegin},
                                                        {token_t::ErrorBegin},
                                                        {token_t::ErrorBegin},
                                                        {token_t::StringBegin},
                                                        {token_t::ErrorBegin},
                                                        {token_t::ErrorBegin},
                                                        {token_t::ErrorBegin},
                                                        {},
                                                        {token_t::ValueBegin}}};
  pda_tlt[static_cast<StateT>(pda_state_t::PD_BOA)] = {{{token_t::ErrorBegin},
                                                        {token_t::ErrorBegin},
                                                        {token_t::ErrorBegin},
                                                        {token_t::ErrorBegin},
                                                        {token_t::ErrorBegin},
                                                        {token_t::ErrorBegin},
                                                        {token_t::ErrorBegin},
                                                        {token_t::ErrorBegin},
                                                        {token_t::ErrorBegin},
                                                        {token_t::ErrorBegin},
                                                        {token_t::StructBegin},
                                                        {token_t::ListBegin},
                                                        {token_t::ErrorBegin},
                                                        {token_t::ListEnd},
                                                        {token_t::StringBegin},
                                                        {token_t::ErrorBegin},
                                                        {token_t::ErrorBegin},
                                                        {token_t::ErrorBegin},
                                                        {},
                                                        {token_t::ValueBegin},
                                                        {token_t::ErrorBegin},
                                                        {token_t::ErrorBegin},
                                                        {token_t::StructEnd},
                                                        {token_t::ErrorBegin},
                                                        {token_t::FieldNameBegin},
                                                        {token_t::ErrorBegin},
                                                        {token_t::ErrorBegin},
                                                        {token_t::ErrorBegin},
                                                        {},
                                                        {token_t::ErrorBegin}}};
  pda_tlt[static_cast<StateT>(pda_state_t::PD_LON)] = {{{token_t::ErrorBegin},
                                                        {token_t::ErrorBegin},
                                                        {token_t::ErrorBegin},
                                                        {token_t::ErrorBegin},
                                                        {token_t::ErrorBegin},
                                                        {token_t::ErrorBegin},
                                                        {token_t::ErrorBegin},
                                                        {token_t::ErrorBegin},
                                                        {token_t::ValueEnd},
                                                        {},
                                                        {token_t::ErrorBegin},
                                                        {token_t::ErrorBegin},
                                                        {token_t::ErrorBegin},
                                                        {token_t::ValueEnd, token_t::ListEnd},
                                                        {token_t::ErrorBegin},
                                                        {token_t::ErrorBegin},
                                                        {token_t::ValueEnd},
                                                        {token_t::ErrorBegin},
                                                        {token_t::ValueEnd},
                                                        {},
                                                        {token_t::ErrorBegin},
                                                        {token_t::ErrorBegin},
                                                        {token_t::ValueEnd, token_t::StructEnd},
                                                        {token_t::ErrorBegin},
                                                        {token_t::ErrorBegin},
                                                        {token_t::ErrorBegin},
                                                        {token_t::ValueEnd},
                                                        {token_t::ErrorBegin},
                                                        {token_t::ValueEnd},
                                                        {}}};
  pda_tlt[static_cast<StateT>(pda_state_t::PD_STR)] = {
    {{}, {}, {}, {}, {token_t::StringEnd}, {}, {}, {}, {}, {},
     {}, {}, {}, {}, {token_t::StringEnd}, {}, {}, {}, {}, {},
     {}, {}, {}, {}, {token_t::StringEnd}, {}, {}, {}, {}, {}}};
  pda_tlt[static_cast<StateT>(pda_state_t::PD_SCE)] = {{{}, {}, {}, {}, {}, {}, {}, {}, {}, {},
                                                        {}, {}, {}, {}, {}, {}, {}, {}, {}, {},
                                                        {}, {}, {}, {}, {}, {}, {}, {}, {}, {}}};
  pda_tlt[static_cast<StateT>(pda_state_t::PD_PVL)] = {{{token_t::ErrorBegin},
                                                        {token_t::ErrorBegin},
                                                        {token_t::ErrorBegin},
                                                        {token_t::ErrorBegin},
                                                        {token_t::ErrorBegin},
                                                        {token_t::ErrorBegin},
                                                        {token_t::ErrorBegin},
                                                        {token_t::ErrorBegin},
                                                        {},
                                                        {token_t::ErrorBegin},
                                                        {token_t::ErrorBegin},
                                                        {token_t::ErrorBegin},
                                                        {token_t::ErrorBegin},
                                                        {token_t::ListEnd},
                                                        {token_t::ErrorBegin},
                                                        {token_t::ErrorBegin},
                                                        {},
                                                        {token_t::ErrorBegin},
                                                        {},
                                                        {token_t::ErrorBegin},
                                                        {token_t::ErrorBegin},
                                                        {token_t::ErrorBegin},
                                                        {token_t::StructEnd},
                                                        {token_t::ErrorBegin},
                                                        {token_t::ErrorBegin},
                                                        {token_t::ErrorBegin},
                                                        {},
                                                        {token_t::ErrorBegin},
                                                        {},
                                                        {token_t::ErrorBegin}}};
  pda_tlt[static_cast<StateT>(pda_state_t::PD_BFN)] = {{{token_t::ErrorBegin},
                                                        {token_t::ErrorBegin},
                                                        {token_t::ErrorBegin},
                                                        {token_t::ErrorBegin},
                                                        {token_t::ErrorBegin},
                                                        {token_t::ErrorBegin},
                                                        {token_t::ErrorBegin},
                                                        {token_t::ErrorBegin},
                                                        {token_t::ErrorBegin},
                                                        {token_t::ErrorBegin},
                                                        {token_t::ErrorBegin},
                                                        {token_t::ErrorBegin},
                                                        {token_t::ErrorBegin},
                                                        {token_t::ErrorBegin},
                                                        {token_t::ErrorBegin},
                                                        {token_t::ErrorBegin},
                                                        {token_t::ErrorBegin},
                                                        {token_t::ErrorBegin},
                                                        {token_t::ErrorBegin},
                                                        {token_t::ErrorBegin},
                                                        {token_t::ErrorBegin},
                                                        {token_t::ErrorBegin},
                                                        {token_t::ErrorBegin},
                                                        {token_t::ErrorBegin},
                                                        {token_t::FieldNameBegin},
                                                        {token_t::ErrorBegin},
                                                        {token_t::ErrorBegin},
                                                        {token_t::ErrorBegin},
                                                        {},
                                                        {token_t::ErrorBegin}}};
  pda_tlt[static_cast<StateT>(pda_state_t::PD_FLN)] = {{{token_t::ErrorBegin},
                                                        {token_t::ErrorBegin},
                                                        {token_t::ErrorBegin},
                                                        {token_t::ErrorBegin},
                                                        {token_t::ErrorBegin},
                                                        {token_t::ErrorBegin},
                                                        {token_t::ErrorBegin},
                                                        {token_t::ErrorBegin},
                                                        {token_t::ErrorBegin},
                                                        {token_t::ErrorBegin},
                                                        {token_t::ErrorBegin},
                                                        {token_t::ErrorBegin},
                                                        {token_t::ErrorBegin},
                                                        {token_t::ErrorBegin},
                                                        {token_t::ErrorBegin},
                                                        {token_t::ErrorBegin},
                                                        {token_t::ErrorBegin},
                                                        {token_t::ErrorBegin},
                                                        {token_t::ErrorBegin},
                                                        {token_t::ErrorBegin},
                                                        {},
                                                        {},
                                                        {},
                                                        {},
                                                        {token_t::FieldNameEnd},
                                                        {},
                                                        {},
                                                        {},
                                                        {},
                                                        {}}};
  pda_tlt[static_cast<StateT>(pda_state_t::PD_FNE)] = {{{token_t::ErrorBegin},
                                                        {token_t::ErrorBegin},
                                                        {token_t::ErrorBegin},
                                                        {token_t::ErrorBegin},
                                                        {token_t::ErrorBegin},
                                                        {token_t::ErrorBegin},
                                                        {token_t::ErrorBegin},
                                                        {token_t::ErrorBegin},
                                                        {token_t::ErrorBegin},
                                                        {token_t::ErrorBegin},
                                                        {token_t::ErrorBegin},
                                                        {token_t::ErrorBegin},
                                                        {token_t::ErrorBegin},
                                                        {token_t::ErrorBegin},
                                                        {token_t::ErrorBegin},
                                                        {token_t::ErrorBegin},
                                                        {token_t::ErrorBegin},
                                                        {token_t::ErrorBegin},
                                                        {token_t::ErrorBegin},
                                                        {token_t::ErrorBegin},
                                                        {},
                                                        {},
                                                        {},
                                                        {},
                                                        {},
                                                        {},
                                                        {},
                                                        {},
                                                        {},
                                                        {}}};
  pda_tlt[static_cast<StateT>(pda_state_t::PD_PFN)] = {{{token_t::ErrorBegin},
                                                        {token_t::ErrorBegin},
                                                        {token_t::ErrorBegin},
                                                        {token_t::ErrorBegin},
                                                        {token_t::ErrorBegin},
                                                        {token_t::ErrorBegin},
                                                        {token_t::ErrorBegin},
                                                        {token_t::ErrorBegin},
                                                        {token_t::ErrorBegin},
                                                        {token_t::ErrorBegin},
                                                        {token_t::ErrorBegin},
                                                        {token_t::ErrorBegin},
                                                        {token_t::ErrorBegin},
                                                        {token_t::ErrorBegin},
                                                        {token_t::ErrorBegin},
                                                        {token_t::ErrorBegin},
                                                        {token_t::ErrorBegin},
                                                        {token_t::ErrorBegin},
                                                        {token_t::ErrorBegin},
                                                        {token_t::ErrorBegin},
                                                        {token_t::ErrorBegin},
                                                        {token_t::ErrorBegin},
                                                        {token_t::ErrorBegin},
                                                        {token_t::ErrorBegin},
                                                        {token_t::ErrorBegin},
                                                        {token_t::ErrorBegin},
                                                        {token_t::ErrorBegin},
                                                        {},
                                                        {},
                                                        {token_t::ErrorBegin}}};
  pda_tlt[static_cast<StateT>(pda_state_t::PD_ERR)] = {{{}, {}, {}, {}, {}, {}, {}, {}, {}, {},
                                                        {}, {}, {}, {}, {}, {}, {}, {}, {}, {},
                                                        {}, {}, {}, {}, {}, {}, {}, {}, {}, {}}};
  return pda_tlt;
}

}  // namespace tokenizer_pda

/**
 * @brief Function object used to filter for brackets and braces that represent push and pop
 * operations
 */
struct JSONToStackOp {
  template <typename StackSymbolT>
  constexpr CUDF_HOST_DEVICE fst::stack_op_type operator()(StackSymbolT const& stack_symbol) const
  {
    return (stack_symbol == '{' || stack_symbol == '[')   ? fst::stack_op_type::PUSH
           : (stack_symbol == '}' || stack_symbol == ']') ? fst::stack_op_type::POP
                                                          : fst::stack_op_type::READ;
  }
};

namespace detail {

void get_stack_context(device_span<SymbolT const> json_in,
                       SymbolT* d_top_of_stack,
                       rmm::cuda_stream_view stream)
{
  constexpr std::size_t single_item = 1;

  // Symbol representing the JSON-root (i.e., we're at nesting level '0')
  constexpr StackSymbolT root_symbol = '_';
  // This can be any stack symbol from the stack alphabet that does not push onto stack
  constexpr StackSymbolT read_symbol = 'x';

  // Number of stack operations in the input (i.e., number of '{', '}', '[', ']' outside of quotes)
  hostdevice_vector<SymbolOffsetT> num_stack_ops(single_item, stream);

  // Sequence of stack symbols and their position in the original input (sparse representation)
  rmm::device_uvector<StackSymbolT> stack_ops{json_in.size(), stream};
  rmm::device_uvector<SymbolOffsetT> stack_op_indices{json_in.size(), stream};

  // Prepare finite-state transducer that only selects '{', '}', '[', ']' outside of quotes
  using ToStackOpFstT =
    cudf::io::fst::detail::Dfa<StackSymbolT,
                               static_cast<int32_t>(
                                 to_stack_op::dfa_symbol_group_id::NUM_SYMBOL_GROUPS),
                               static_cast<int32_t>(to_stack_op::dfa_states::TT_NUM_STATES)>;
  ToStackOpFstT json_to_stack_ops_fst{to_stack_op::symbol_groups,
                                      to_stack_op::transition_table,
                                      to_stack_op::translation_table,
                                      stream};

  // "Search" for relevant occurrence of brackets and braces that indicate the beginning/end
  // of structs/lists
  json_to_stack_ops_fst.Transduce(json_in.begin(),
                                  static_cast<SymbolOffsetT>(json_in.size()),
                                  stack_ops.data(),
                                  stack_op_indices.data(),
                                  num_stack_ops.device_ptr(),
                                  to_stack_op::start_state,
                                  stream);

  // stack operations with indices are converted to top of the stack for each character in the input
  fst::sparse_stack_op_to_top_of_stack<StackLevelT>(
    stack_ops.data(),
    device_span<SymbolOffsetT>{stack_op_indices.data(), stack_op_indices.size()},
    JSONToStackOp{},
    d_top_of_stack,
    root_symbol,
    read_symbol,
    json_in.size(),
    stream);
}

// TODO: return pair of device_uvector instead of passing pre-allocated pointers.
void get_token_stream(device_span<SymbolT const> json_in,
                      PdaTokenT* d_tokens,
                      SymbolOffsetT* d_tokens_indices,
                      SymbolOffsetT* d_num_written_tokens,
                      rmm::cuda_stream_view stream)
{
  // Memory holding the top-of-stack stack context for the input
  rmm::device_uvector<StackSymbolT> stack_op_indices{json_in.size(), stream};

  // Identify what is the stack context for each input character (is it: JSON-root, struct, or list)
  get_stack_context(json_in, stack_op_indices.data(), stream);

  // Prepare for PDA transducer pass, merging input symbols with stack symbols
  rmm::device_uvector<PdaSymbolGroupIdT> pda_sgids{json_in.size(), stream};
  auto zip_in = thrust::make_zip_iterator(json_in.data(), stack_op_indices.data());
  thrust::transform(rmm::exec_policy(stream),
                    zip_in,
                    zip_in + json_in.size(),
                    pda_sgids.data(),
                    tokenizer_pda::PdaSymbolToSymbolGroupId{});

  // PDA transducer alias
  using ToTokenStreamFstT =
    cudf::io::fst::detail::Dfa<StackSymbolT,
                               tokenizer_pda::NUM_PDA_SGIDS,
                               static_cast<tokenizer_pda::StateT>(
                                 tokenizer_pda::pda_state_t::PD_NUM_STATES)>;

  // Instantiating PDA transducer
  ToTokenStreamFstT json_to_tokens_fst{tokenizer_pda::pda_sgids,
                                       tokenizer_pda::get_transition_table(),
                                       tokenizer_pda::get_translation_table(),
                                       stream};

  // Perform a PDA-transducer pass
  json_to_tokens_fst.Transduce(pda_sgids.begin(),
                               static_cast<SymbolOffsetT>(json_in.size()),
                               d_tokens,
                               d_tokens_indices,
                               d_num_written_tokens,
                               tokenizer_pda::start_state,
                               stream);
}

/**
 * @brief Parses the given JSON string and generates a tree representation of the given input.
 *
 * @param[in,out] root_column The root column of the hierarchy of columns into which data is parsed
 * @param[in,out] current_data_path The stack represents the path from the JSON root node to the
 * first node encountered in \p input
 * @param[in] input The JSON input in host memory
 * @param[in] d_input The JSON input in device memory
 * @param[in] stream The CUDA stream to which kernels are dispatched
 * @return The columnar representation of the data from the given JSON input
 */
void make_json_column(json_column& root_column,
                      std::stack<tree_node>& current_data_path,
                      host_span<SymbolT const> input,
                      device_span<SymbolT const> d_input,
                      rmm::cuda_stream_view stream)
{
  // Default name for a list's child column
  std::string const list_child_name = "element";

  constexpr std::size_t single_item = 1;
  hostdevice_vector<PdaTokenT> tokens_gpu{input.size(), stream};
  hostdevice_vector<SymbolOffsetT> token_indices_gpu{input.size(), stream};
  hostdevice_vector<SymbolOffsetT> num_tokens_out{single_item, stream};

  // Parse the JSON and get the token stream
  get_token_stream(d_input,
                   tokens_gpu.device_ptr(),
                   token_indices_gpu.device_ptr(),
                   num_tokens_out.device_ptr(),
                   stream);

  // Copy the JSON tokens to the host
  token_indices_gpu.device_to_host(stream);
  tokens_gpu.device_to_host(stream);
  num_tokens_out.device_to_host(stream);

  // Make sure tokens have been copied to the host
  stream.synchronize();

  // Whether this token is the valid token to begin the JSON document with
  auto is_valid_root_token = [](PdaTokenT const token) {
    switch (token) {
      case token_t::StructBegin:
      case token_t::ListBegin:
      case token_t::StringBegin:
      case token_t::ValueBegin: return true;
      default: return false;
    };
  };

  // Returns the token's corresponding column type
  auto token_to_column_type = [](PdaTokenT const token) {
    switch (token) {
      case token_t::StructBegin: return json_col_t::StructColumn;
      case token_t::ListBegin: return json_col_t::ListColumn;
      case token_t::StringBegin: return json_col_t::StringColumn;
      case token_t::ValueBegin: return json_col_t::StringColumn;
      default: return json_col_t::Unknown;
    };
  };

  // Whether this token is a beginning-of-list or beginning-of-struct token
  auto is_nested_token = [](PdaTokenT const token) {
    switch (token) {
      case token_t::StructBegin:
      case token_t::ListBegin: return true;
      default: return false;
    };
  };

  // Skips the quote char if the token is a beginning-of-string or beginning-of-field-name token
  auto get_token_index = [](PdaTokenT const token, SymbolOffsetT const token_index) {
    constexpr SymbolOffsetT skip_quote_char = 1;
    switch (token) {
      case token_t::StringBegin: return token_index + skip_quote_char;
      case token_t::FieldNameBegin: return token_index + skip_quote_char;
      default: return token_index;
    };
  };

  // The end-of-* partner token for a given beginning-of-* token
  auto end_of_partner = [](PdaTokenT const token) {
    switch (token) {
      case token_t::StringBegin: return token_t::StringEnd;
      case token_t::ValueBegin: return token_t::ValueEnd;
      case token_t::FieldNameBegin: return token_t::FieldNameEnd;
      default: return token_t::ErrorBegin;
    };
  };

#ifdef NJP_DEBUG_PRINT
  auto column_type_string = [](json_col_t column_type) {
    switch (column_type) {
      case json_col_t::Unknown: return "Unknown";
      case json_col_t::ListColumn: return "List";
      case json_col_t::StructColumn: return "Struct";
      case json_col_t::StringColumn: return "String";
      default: return "Unknown";
    }
  };

  auto token_to_string = [](PdaTokenT token_type) {
    switch (token_type) {
      case token_t::StructBegin: return "StructBegin";
      case token_t::StructEnd: return "StructEnd";
      case token_t::ListBegin: return "ListBegin";
      case token_t::ListEnd: return "ListEnd";
      case token_t::FieldNameBegin: return "FieldNameBegin";
      case token_t::FieldNameEnd: return "FieldNameEnd";
      case token_t::StringBegin: return "StringBegin";
      case token_t::StringEnd: return "StringEnd";
      case token_t::ValueBegin: return "ValueBegin";
      case token_t::ValueEnd: return "ValueEnd";
      case token_t::ErrorBegin: return "ErrorBegin";
      default: return "Unknown";
    }
  };
#endif

  /**
   * @brief Updates the given row in the given column with a new string_end and child_count. In
   * particular, updating the child count is relevant for list columns.
   */
  auto update_row =
    [](json_column* column, uint32_t row_index, uint32_t string_end, uint32_t child_count) {
#ifdef NJP_DEBUG_PRINT
      std::cout << "  -> update_row()\n";
      std::cout << "  ---> col@" << column << "\n";
      std::cout << "  ---> row #" << row_index << "\n";
      std::cout << "  ---> string_lengths = " << (string_end - column->string_offsets[row_index])
                << "\n";
      std::cout << "  ---> child_offsets = " << (column->child_offsets[row_index + 1] + child_count)
                << "\n";
#endif
      column->string_lengths[row_index]    = column->child_offsets[row_index + 1] + child_count;
      column->child_offsets[row_index + 1] = column->child_offsets[row_index + 1] + child_count;
    };

  /**
   * @brief Gets the currently selected child column given a \p current_data_path.
   *
   * That is, if \p current_data_path top-of-stack is
   * (a) a struct, the selected child column corresponds to the child column of the last field name
   * node encountered.
   * (b) a list, the selected child column corresponds to single child column of
   * the list column. In this case, the child column may not exist yet.
   */
  auto get_selected_column = [&list_child_name](std::stack<tree_node>& current_data_path) {
    json_column* selected_col = current_data_path.top().current_selected_col;

    // If the node does not have a selected column yet
    if (selected_col == nullptr) {
      // We're looking at the child column of a list column
      if (current_data_path.top().column->type == json_col_t::ListColumn) {
        CUDF_EXPECTS(current_data_path.top().column->child_columns.size() <= 1,
                     "Encountered a list column with more than a single child column");
        // The child column has yet to be created
        if (current_data_path.top().column->child_columns.size() == 0) {
          current_data_path.top().column->child_columns.emplace(std::string{list_child_name},
                                                                json_column{json_col_t::Unknown});
        }
        current_data_path.top().current_selected_col =
          &current_data_path.top().column->child_columns.begin()->second;
        selected_col = current_data_path.top().current_selected_col;
      } else {
        CUDF_FAIL("Trying to retrieve child column without encountering a field name.");
      }
    }
#ifdef NJP_DEBUG_PRINT
    std::cout << "  -> get_selected_column()\n";
    std::cout << "  ---> selected col@" << selected_col << "\n";
#endif
    return selected_col;
  };

  /**
   * @brief Returns a pointer to the child column with the given \p field_name within the current
   * struct column.
   */
  auto select_column = [](std::stack<tree_node>& current_data_path, std::string const& field_name) {
#ifdef NJP_DEBUG_PRINT
    std::cout << "  -> select_column(" << field_name << ")\n";
#endif
    // The field name's parent struct node
    auto& current_struct_node = current_data_path.top();

    // Verify that the field name node is actually a child of a struct
    CUDF_EXPECTS(current_data_path.top().column->type == json_col_t::StructColumn,
                 "Invalid JSON token sequence");

    json_column* struct_col  = current_struct_node.column;
    auto const& child_col_it = struct_col->child_columns.find(field_name);

    // The field name's column exists already, select that as the struct node's currently selected
    // child column
    if (child_col_it != struct_col->child_columns.end()) { return &child_col_it->second; }

    // The field name's column does not exist yet, so we have to append the child column to the
    // struct column
    return &struct_col->child_columns.emplace(field_name, json_column{}).first->second;
  };

  /**
   * @brief Gets the row offset at which to insert. I.e., for a child column of a list column, we
   * just have to append the row to the end. Otherwise we have to propagate the row offset from the
   * parent struct column.
   */
  auto get_target_row_index = [](std::stack<tree_node> const& current_data_path,
                                 json_column* target_column) {
#ifdef NJP_DEBUG_PRINT
    std::cout << " -> target row: "
              << ((current_data_path.top().column->type == json_col_t::ListColumn)
                    ? target_column->current_offset
                    : current_data_path.top().row_index)
              << "\n";
#endif
    return (current_data_path.top().column->type == json_col_t::ListColumn)
             ? target_column->current_offset
             : current_data_path.top().row_index;
  };

  // The offset of the token currently being processed
  std::size_t offset = 0;

  // Giving names to magic constants
  constexpr uint32_t row_offset_zero  = 0;
  constexpr uint32_t zero_child_count = 0;

  //--------------------------------------------------------------------------------
  // INITIALIZE JSON ROOT NODE
  //--------------------------------------------------------------------------------
  // The JSON root may only be a struct, list, string, or value node
  CUDF_EXPECTS(num_tokens_out[0] > 0, "Empty JSON input not supported");
  CUDF_EXPECTS(is_valid_root_token(tokens_gpu[offset]), "Invalid beginning of JSON document");

  // The JSON root is either a struct or list
  if (is_nested_token(tokens_gpu[offset])) {
    // Initialize the root column and append this row to it
    root_column.append_row(row_offset_zero,
                           token_to_column_type(tokens_gpu[offset]),
                           get_token_index(tokens_gpu[offset], token_indices_gpu[offset]),
                           get_token_index(tokens_gpu[offset], token_indices_gpu[offset]),
                           0);

    // Push the root node onto the stack for the data path
    current_data_path.push({&root_column, row_offset_zero, nullptr, zero_child_count});

    // Continue with the next token from the token stream
    offset++;
  }
  // The JSON is a simple scalar value -> create simple table and return
  else {
    constexpr SymbolOffsetT max_tokens_for_scalar_value = 2;
    CUDF_EXPECTS(num_tokens_out[0] <= max_tokens_for_scalar_value,
                 "Invalid JSON format. Expected just a scalar value.");

    // If this isn't the only token, verify the subsequent token is the correct end-of-* partner
    if ((offset + 1) < num_tokens_out[0]) {
      CUDF_EXPECTS(tokens_gpu[offset + 1] == end_of_partner(tokens_gpu[offset]),
                   "Invalid JSON token sequence");
    }

    // The offset to the first symbol from the JSON input associated with the current token
    auto const& token_begin_offset = get_token_index(tokens_gpu[offset], token_indices_gpu[offset]);

    // The offset to one past the last symbol associated with the current token
    // Literals without trailing space are missing the corresponding end-of-* counterpart.
    auto const& token_end_offset =
      (offset + 1 < num_tokens_out[0])
        ? get_token_index(tokens_gpu[offset + 1], token_indices_gpu[offset + 1])
        : input.size();

    root_column.append_row(row_offset_zero,
                           json_col_t::StringColumn,
                           token_begin_offset,
                           token_end_offset,
                           zero_child_count);
    return;
  }

  while (offset < num_tokens_out[0]) {
    // Verify there's at least the JSON root node left on the stack to which we can append data
    CUDF_EXPECTS(current_data_path.size() > 0, "Invalid JSON structure");

    // Verify that the current node in the tree (which becomes this nodes parent) can have children
    CUDF_EXPECTS(current_data_path.top().column->type == json_col_t::ListColumn or
                   current_data_path.top().column->type == json_col_t::StructColumn,
                 "Invalid JSON structure");

    // The token we're currently parsing
    auto const& token = tokens_gpu[offset];

#ifdef NJP_DEBUG_PRINT
    std::cout << "[" << token_to_string(token) << "]\n";
#endif

    // StructBegin token
    if (token == token_t::StructBegin) {
      // Get this node's column. That is, the parent node's selected column:
      // (a) if parent is a list, then this will (create and) return the list's only child column
      // (b) if parent is a struct, then this will return the column selected by the last field name
      // encountered.
      json_column* selected_col = get_selected_column(current_data_path);

      // Get the row offset at which to insert
      auto const target_row_index = get_target_row_index(current_data_path, selected_col);

      // Increment parent's child count and insert this struct node into the data path
      current_data_path.top().num_children++;
      current_data_path.push({selected_col, target_row_index, nullptr, zero_child_count});

      // Add this struct node to the current column
      selected_col->append_row(target_row_index,
                               token_to_column_type(tokens_gpu[offset]),
                               get_token_index(tokens_gpu[offset], token_indices_gpu[offset]),
                               get_token_index(tokens_gpu[offset], token_indices_gpu[offset]),
                               zero_child_count);
    }

    // StructEnd token
    else if (token == token_t::StructEnd) {
      // Verify that this node in fact a struct node (i.e., it was part of a struct column)
      CUDF_EXPECTS(current_data_path.top().column->type == json_col_t::StructColumn,
                   "Broken invariant while parsing JSON");
      CUDF_EXPECTS(current_data_path.top().column != nullptr,
                   "Broken invariant while parsing JSON");

      // Update row to account for string offset
      update_row(current_data_path.top().column,
                 current_data_path.top().row_index,
                 get_token_index(tokens_gpu[offset], token_indices_gpu[offset]),
                 current_data_path.top().num_children);

      // Pop struct from the path stack
      current_data_path.pop();
    }

    // ListBegin token
    else if (token == token_t::ListBegin) {
      // Get the selected column
      json_column* selected_col = get_selected_column(current_data_path);

      // Get the row offset at which to insert
      auto const target_row_index = get_target_row_index(current_data_path, selected_col);

      // Increment parent's child count and insert this struct node into the data path
      current_data_path.top().num_children++;
      current_data_path.push({selected_col, target_row_index, nullptr, zero_child_count});

      // Add this struct node to the current column
      selected_col->append_row(target_row_index,
                               token_to_column_type(tokens_gpu[offset]),
                               get_token_index(tokens_gpu[offset], token_indices_gpu[offset]),
                               get_token_index(tokens_gpu[offset], token_indices_gpu[offset]),
                               zero_child_count);
    }

    // ListEnd token
    else if (token == token_t::ListEnd) {
      // Verify that this node in fact a list node (i.e., it was part of a list column)
      CUDF_EXPECTS(current_data_path.top().column->type == json_col_t::ListColumn,
                   "Broken invariant while parsing JSON");
      CUDF_EXPECTS(current_data_path.top().column != nullptr,
                   "Broken invariant while parsing JSON");

      // Update row to account for string offset
      update_row(current_data_path.top().column,
                 current_data_path.top().row_index,
                 get_token_index(tokens_gpu[offset], token_indices_gpu[offset]),
                 current_data_path.top().num_children);

      // Pop list from the path stack
      current_data_path.pop();
    }

    // Error token
    else if (token == token_t::ErrorBegin) {
#ifdef NJP_DEBUG_PRINT
      std::cout << "[ErrorBegin]\n";
#endif
      CUDF_FAIL("Parser encountered an invalid format.");
    }

    // FieldName, String, or Value (begin, end)-pair
    else if (token == token_t::FieldNameBegin or token == token_t::StringBegin or
             token == token_t::ValueBegin) {
      // Verify that this token has the right successor to build a correct (being, end) token pair
      CUDF_EXPECTS((offset + 1) < num_tokens_out[0], "Invalid JSON token sequence");
      CUDF_EXPECTS(tokens_gpu[offset + 1] == end_of_partner(token), "Invalid JSON token sequence");

      // The offset to the first symbol from the JSON input associated with the current token
      auto const& token_begin_offset =
        get_token_index(tokens_gpu[offset], token_indices_gpu[offset]);

      // The offset to one past the last symbol associated with the current token
      auto const& token_end_offset =
        get_token_index(tokens_gpu[offset + 1], token_indices_gpu[offset + 1]);

      // FieldNameBegin
      // For the current struct node in the tree, select the child column corresponding to this
      // field name
      if (token == token_t::FieldNameBegin) {
        std::string field_name{input.data() + token_begin_offset,
                               (token_end_offset - token_begin_offset)};
        current_data_path.top().current_selected_col = select_column(current_data_path, field_name);
      }
      // StringBegin
      // ValueBegin
      // As we currently parse to string columns there's no further differentiation
      else if (token == token_t::StringBegin or token == token_t::ValueBegin) {
        // Get the selected column
        json_column* selected_col = get_selected_column(current_data_path);

        // Get the row offset at which to insert
        auto const target_row_index = get_target_row_index(current_data_path, selected_col);

        current_data_path.top().num_children++;

        selected_col->append_row(target_row_index,
                                 token_to_column_type(token),
                                 token_begin_offset,
                                 token_end_offset,
                                 zero_child_count);
      } else {
        CUDF_FAIL("Unknown JSON token");
      }

      // As we've also consumed the end-of-* token, we advance the processed token offset by one
      offset++;
    }

    offset++;
  }

  // Make sure all of a struct's child columns have the same length
  root_column.level_child_cols_recursively(root_column.current_offset);
}

std::pair<std::unique_ptr<column>, std::vector<column_name_info>> json_column_to_cudf_column(
  json_column const& json_col,
  device_span<SymbolT const> d_input,
  rmm::cuda_stream_view stream,
  rmm::mr::device_memory_resource* mr)
{
  auto make_validity =
    [stream, mr](json_column const& json_col) -> std::pair<rmm::device_buffer, size_type> {
    if (json_col.current_offset == json_col.valid_count) { return {rmm::device_buffer{}, 0}; }

    return {rmm::device_buffer{json_col.validity.data(),
                               bitmask_allocation_size_bytes(json_col.current_offset),
                               stream,
                               mr},
            json_col.current_offset - json_col.valid_count};
  };

  switch (json_col.type) {
    case json_col_t::StringColumn: {
      // move string_offsets to GPU and transform to string column
      auto const col_size      = json_col.string_offsets.size();
      using char_length_pair_t = thrust::pair<const char*, size_type>;
      CUDF_EXPECTS(json_col.string_offsets.size() == json_col.string_lengths.size(),
                   "string offset, string length mismatch");
      rmm::device_uvector<char_length_pair_t> d_string_data(col_size, stream);
      rmm::device_uvector<json_column::row_offset_t> d_string_offsets =
        cudf::detail::make_device_uvector_async(json_col.string_offsets, stream);
      rmm::device_uvector<json_column::row_offset_t> d_string_lengths =
        cudf::detail::make_device_uvector_async(json_col.string_lengths, stream);
      auto offset_length_it =
        thrust::make_zip_iterator(d_string_offsets.begin(), d_string_lengths.begin());
      thrust::transform(rmm::exec_policy(stream),
                        offset_length_it,
                        offset_length_it + col_size,
                        d_string_data.data(),
                        [data = d_input.data()] __device__(auto ip) {
                          return char_length_pair_t{data + thrust::get<0>(ip), thrust::get<1>(ip)};
                        });
      auto str_col_ptr                  = make_strings_column(d_string_data, stream, mr);
      auto [result_bitmask, null_count] = make_validity(json_col);
      str_col_ptr->set_null_mask(result_bitmask, null_count);
      return {std::move(str_col_ptr), {{"offsets"}, {"chars"}}};
      break;
    }
    case json_col_t::StructColumn: {
      std::vector<std::unique_ptr<column>> child_columns;
      std::vector<column_name_info> column_names{};
      size_type num_rows{json_col.current_offset};
      // Create children columns
      for (auto const& col : json_col.child_columns) {
        column_names.emplace_back(col.first);
        auto const& child_col      = col.second;
        auto [child_column, names] = json_column_to_cudf_column(child_col, d_input, stream, mr);
        CUDF_EXPECTS(num_rows == child_column->size(),
                     "All children columns must have the same size");
        child_columns.push_back(std::move(child_column));
        column_names.back().children = names;
      }
      auto [result_bitmask, null_count] = make_validity(json_col);
      return {
        make_structs_column(
          num_rows, std::move(child_columns), null_count, std::move(result_bitmask), stream, mr),
        column_names};
      break;
    }
    case json_col_t::ListColumn: {
      size_type num_rows = json_col.child_offsets.size();
      std::vector<column_name_info> column_names{};
      column_names.emplace_back("offsets");
      column_names.emplace_back(json_col.child_columns.begin()->first);

      rmm::device_uvector<json_column::row_offset_t> d_offsets =
        cudf::detail::make_device_uvector_async(json_col.child_offsets, stream, mr);
      auto offsets_column =
        std::make_unique<column>(data_type{type_id::INT32}, num_rows, d_offsets.release());
      // Create children column
      auto [child_column, names] =
        json_column_to_cudf_column(json_col.child_columns.begin()->second, d_input, stream, mr);
      column_names.back().children      = names;
      auto [result_bitmask, null_count] = make_validity(json_col);
      return {make_lists_column(num_rows - 1,
                                std::move(offsets_column),
                                std::move(child_column),
                                null_count,
                                std::move(result_bitmask),
                                stream,
                                mr),
              std::move(column_names)};
      break;
    }
    default: CUDF_FAIL("Unsupported column type, yet to be implemented"); break;
  }

  return {};
}

table_with_metadata parse_nested_json(host_span<SymbolT const> input,
                                      rmm::cuda_stream_view stream,
                                      rmm::mr::device_memory_resource* mr)
{
  // Allocate device memory for the JSON input & copy over to device
  rmm::device_uvector<SymbolT> d_input = cudf::detail::make_device_uvector_async(input, stream);

  // Get internal JSON column
  json_column root_column{};
  std::stack<tree_node> data_path{};
  make_json_column(root_column, data_path, input, d_input, stream);

  // Verify that we were in fact given a list of structs (or in JSON speech: an array of objects)
  auto constexpr single_child_col_count = 1;
  CUDF_EXPECTS(root_column.type == json_col_t::ListColumn and
                 root_column.child_columns.size() == single_child_col_count and
                 root_column.child_columns.begin()->second.type == json_col_t::StructColumn,
               "Currently the nested JSON parser only supports an array of (nested) objects");

  // Slice off the root list column, which has only a single row that contains all the structs
  auto const& root_struct_col = root_column.child_columns.begin()->second;

  // Initialize meta data to be populated while recursing through the tree of columns
  std::vector<std::unique_ptr<column>> out_columns;
  std::vector<column_name_info> out_column_names;

  // Iterate over the struct's child columns and convert to cudf column
  for (auto const& [col_name, json_col] : root_struct_col.child_columns) {
    // Insert this columns name into the schema
    out_column_names.emplace_back(col_name);

    // Get this JSON column's cudf column and schema info
    auto [cudf_col, col_name_info]   = json_column_to_cudf_column(json_col, d_input, stream, mr);
    out_column_names.back().children = std::move(col_name_info);
    out_columns.emplace_back(std::move(cudf_col));
  }

  return table_with_metadata{std::make_unique<table>(std::move(out_columns)),
                             {{}, out_column_names}};
}

}  // namespace detail
}  // namespace cudf::io::json

// Debug print flag
#undef NJP_DEBUG_PRINT
