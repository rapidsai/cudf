#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2022, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "nested_json.hpp"

#include <io/fst/logical_stack.cuh>
#include <io/fst/lookup_tables.cuh>
#include <io/utilities/parsing_utils.cuh>
#include <io/utilities/type_inference.cuh>

#include <cudf/column/column_factories.hpp>
#include <cudf/detail/nvtx/ranges.hpp>
#include <cudf/detail/utilities/vector_factories.hpp>
#include <cudf/detail/utilities/visitor_overload.hpp>
#include <cudf/detail/valid_if.cuh>
#include <cudf/io/detail/data_casting.cuh>
#include <cudf/io/json.hpp>
#include <cudf/table/table.hpp>
#include <cudf/types.hpp>
#include <cudf/utilities/bit.hpp>
#include <cudf/utilities/error.hpp>
#include <cudf/utilities/span.hpp>

#include <rmm/device_scalar.hpp>
#include <rmm/device_uvector.hpp>
#include <rmm/exec_policy.hpp>

#include <thrust/device_vector.h>
#include <thrust/iterator/transform_iterator.h>
#include <thrust/transform.h>

#include <stack>

// Debug print flag
#ifndef NJP_DEBUG_PRINT
//#define NJP_DEBUG_PRINT
#endif

namespace {

/**
 * @brief While parsing the token stream, we use a stack of tree_nodes to maintain all the
 * information about the data path that is relevant.
 */
struct tree_node {
  // The column that this node is associated with
  cudf::io::json::json_column* column;

  // The row offset that this node belongs to within the given column
  uint32_t row_index;

  // Selected child column
  // E.g., if this is a struct node, and we subsequently encountered the field name "a", then this
  // point's to the struct's "a" child column
  cudf::io::json::json_column* current_selected_col = nullptr;

  std::size_t num_children = 0;
};
}  // namespace

namespace cudf::io::json {

// JSON to stack operator DFA (Deterministic Finite Automata)
namespace to_stack_op {

// Type used to represent the target state in the transition table
using StateT = char;

/**
 * @brief Definition of the DFA's states
 */
enum class dfa_states : StateT {
  // The active state while outside of a string. When encountering an opening bracket or curly
  // brace, we push it onto the stack. When encountering a closing bracket or brace, we pop from the
  // stack.
  TT_OOS = 0U,

  // The active state while within a string (e.g., field name or a string value). We do not push or
  // pop from the stack while in this state.
  TT_STR,

  // The active state after encountering an escape symbol (e.g., '\'), while in the TT_STR state.
  TT_ESC,

  // Total number of states
  TT_NUM_STATES
};

// Aliases for readability of the transition table
constexpr auto TT_OOS = dfa_states::TT_OOS;
constexpr auto TT_STR = dfa_states::TT_STR;
constexpr auto TT_ESC = dfa_states::TT_ESC;

/**
 * @brief Definition of the symbol groups
 */
enum class dfa_symbol_group_id : uint8_t {
  OPENING_BRACE,     ///< Opening brace SG: {
  OPENING_BRACKET,   ///< Opening bracket SG: [
  CLOSING_BRACE,     ///< Closing brace SG: }
  CLOSING_BRACKET,   ///< Closing bracket SG: ]
  QUOTE_CHAR,        ///< Quote character SG: "
  ESCAPE_CHAR,       ///< Escape character SG: '\'
  OTHER_SYMBOLS,     ///< SG implicitly matching all other characters
  NUM_SYMBOL_GROUPS  ///< Total number of symbol groups
};

constexpr auto TT_NUM_STATES     = static_cast<StateT>(dfa_states::TT_NUM_STATES);
constexpr auto NUM_SYMBOL_GROUPS = static_cast<uint32_t>(dfa_symbol_group_id::NUM_SYMBOL_GROUPS);

// The i-th string representing all the characters of a symbol group
std::array<std::string, NUM_SYMBOL_GROUPS - 1> const symbol_groups{
  {{"{"}, {"["}, {"}"}, {"]"}, {"\""}, {"\\"}}};

// Transition table
std::array<std::array<dfa_states, NUM_SYMBOL_GROUPS>, TT_NUM_STATES> const transition_table{
  {/* IN_STATE          {       [       }       ]       "       \    OTHER */
   /* TT_OOS    */ {{TT_OOS, TT_OOS, TT_OOS, TT_OOS, TT_STR, TT_OOS, TT_OOS}},
   /* TT_STR    */ {{TT_STR, TT_STR, TT_STR, TT_STR, TT_OOS, TT_ESC, TT_STR}},
   /* TT_ESC    */ {{TT_STR, TT_STR, TT_STR, TT_STR, TT_STR, TT_STR, TT_STR}}}};

// Translation table (i.e., for each transition, what are the symbols that we output)
std::array<std::array<std::vector<char>, NUM_SYMBOL_GROUPS>, TT_NUM_STATES> const translation_table{
  {/* IN_STATE         {      [      }      ]      "      \    OTHER */
   /* TT_OOS    */ {{{'{'}, {'['}, {'}'}, {']'}, {}, {}, {}}},
   /* TT_STR    */ {{{}, {}, {}, {}, {}, {}, {}}},
   /* TT_ESC    */ {{{}, {}, {}, {}, {}, {}, {}}}}};

// The DFA's starting state
constexpr auto start_state = static_cast<StateT>(TT_OOS);
}  // namespace to_stack_op

// JSON tokenizer pushdown automaton
namespace tokenizer_pda {

// Type used to represent the target state in the transition table
using StateT = char;

/**
 * @brief Symbol groups for the input alphabet for the pushdown automaton
 */
enum class symbol_group_id : PdaSymbolGroupIdT {
  /// Opening brace
  OPENING_BRACE,
  /// Opening bracket
  OPENING_BRACKET,
  /// Closing brace
  CLOSING_BRACE,
  /// Closing bracket
  CLOSING_BRACKET,
  /// Quote
  QUOTE,
  /// Escape
  ESCAPE,
  /// Comma
  COMMA,
  /// Colon
  COLON,
  /// Whitespace
  WHITE_SPACE,
  /// Linebreak
  LINE_BREAK,
  /// Other (any input symbol not assigned to one of the above symbol groups)
  OTHER,
  /// Total number of symbol groups amongst which to differentiate
  NUM_PDA_INPUT_SGS
};

/**
 * @brief Symbols in the stack alphabet
 */
enum class stack_symbol_group_id : PdaStackSymbolGroupIdT {
  /// Symbol representing that we're at the JSON root (nesting level 0)
  STACK_ROOT,

  /// Symbol representing that we're currently within a list object
  STACK_LIST,

  /// Symbol representing that we're currently within a struct object
  STACK_STRUCT,

  /// Total number of symbols in the stack alphabet
  NUM_STACK_SGS
};
constexpr auto NUM_PDA_INPUT_SGS =
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::NUM_PDA_INPUT_SGS);
constexpr auto NUM_STACK_SGS =
  static_cast<PdaStackSymbolGroupIdT>(stack_symbol_group_id::NUM_STACK_SGS);

/// Total number of symbol groups to differentiate amongst (stack alphabet * input alphabet)
constexpr PdaSymbolGroupIdT NUM_PDA_SGIDS = NUM_PDA_INPUT_SGS * NUM_STACK_SGS;

/// Mapping a input symbol to the symbol group id
static __constant__ PdaSymbolGroupIdT tos_sg_to_pda_sgid[] = {
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::OTHER),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::OTHER),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::OTHER),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::OTHER),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::OTHER),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::OTHER),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::OTHER),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::OTHER),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::OTHER),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::WHITE_SPACE),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::LINE_BREAK),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::OTHER),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::OTHER),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::WHITE_SPACE),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::OTHER),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::OTHER),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::OTHER),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::OTHER),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::OTHER),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::OTHER),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::OTHER),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::OTHER),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::OTHER),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::OTHER),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::OTHER),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::OTHER),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::OTHER),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::OTHER),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::OTHER),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::OTHER),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::OTHER),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::OTHER),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::WHITE_SPACE),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::OTHER),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::QUOTE),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::OTHER),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::OTHER),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::OTHER),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::OTHER),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::OTHER),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::OTHER),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::OTHER),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::OTHER),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::OTHER),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::COMMA),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::OTHER),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::OTHER),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::OTHER),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::OTHER),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::OTHER),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::OTHER),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::OTHER),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::OTHER),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::OTHER),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::OTHER),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::OTHER),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::OTHER),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::OTHER),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::COLON),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::OTHER),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::OTHER),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::OTHER),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::OTHER),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::OTHER),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::OTHER),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::OTHER),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::OTHER),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::OTHER),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::OTHER),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::OTHER),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::OTHER),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::OTHER),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::OTHER),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::OTHER),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::OTHER),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::OTHER),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::OTHER),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::OTHER),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::OTHER),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::OTHER),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::OTHER),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::OTHER),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::OTHER),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::OTHER),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::OTHER),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::OTHER),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::OTHER),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::OTHER),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::OTHER),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::OTHER),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::OTHER),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::OPENING_BRACKET),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::ESCAPE),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::CLOSING_BRACKET),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::OTHER),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::OTHER),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::OTHER),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::OTHER),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::OTHER),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::OTHER),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::OTHER),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::OTHER),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::OTHER),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::OTHER),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::OTHER),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::OTHER),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::OTHER),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::OTHER),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::OTHER),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::OTHER),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::OTHER),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::OTHER),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::OTHER),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::OTHER),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::OTHER),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::OTHER),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::OTHER),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::OTHER),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::OTHER),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::OTHER),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::OTHER),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::OTHER),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::OTHER),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::OPENING_BRACE),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::OTHER),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::CLOSING_BRACE),
  static_cast<PdaSymbolGroupIdT>(symbol_group_id::OTHER)};

/**
 * @brief Maps a (top-of-stack symbol, input symbol)-pair to a symbol group id of the deterministic
 * visibly pushdown automaton (DVPA)
 */
struct PdaSymbolToSymbolGroupId {
  template <typename SymbolT, typename StackSymbolT>
  __device__ __forceinline__ PdaSymbolGroupIdT
  operator()(thrust::tuple<SymbolT, StackSymbolT> symbol_pair)
  {
    // The symbol read from the input
    auto symbol = thrust::get<0>(symbol_pair);

    // The stack symbol (i.e., what is on top of the stack at the time the input symbol was read)
    // I.e., whether we're reading in something within a struct, a list, or the JSON root
    auto stack_symbol = thrust::get<1>(symbol_pair);

    // The stack symbol offset: '_' is the root group (0), '[' is the list group (1), '{' is the
    // struct group (2)
    int32_t stack_idx = static_cast<PdaStackSymbolGroupIdT>(
      (stack_symbol == '_') ? stack_symbol_group_id::STACK_ROOT
                            : ((stack_symbol == '[') ? stack_symbol_group_id::STACK_LIST
                                                     : stack_symbol_group_id::STACK_STRUCT));

    // The relative symbol group id of the current input symbol
    constexpr auto pda_sgid_lookup_size =
      static_cast<int32_t>(sizeof(tos_sg_to_pda_sgid) / sizeof(tos_sg_to_pda_sgid[0]));
    PdaSymbolGroupIdT symbol_gid =
      tos_sg_to_pda_sgid[min(static_cast<int32_t>(symbol), pda_sgid_lookup_size - 1)];
    return stack_idx * static_cast<PdaSymbolGroupIdT>(symbol_group_id::NUM_PDA_INPUT_SGS) +
           symbol_gid;
  }
};

// The states defined by the pushdown automaton
enum class pda_state_t : StateT {
  // Beginning of value
  PD_BOV,
  // Beginning of array
  PD_BOA,
  // Literal or number
  PD_LON,
  // String
  PD_STR,
  // After escape char when within string
  PD_SCE,
  // After having parsed a value
  PD_PVL,
  // Before the next field name
  PD_BFN,
  // Field name
  PD_FLN,
  // After escape char when within field name
  PD_FNE,
  // After a field name inside a struct
  PD_PFN,
  // Error state (trap state)
  PD_ERR,
  // Total number of PDA states
  PD_NUM_STATES
};

// Aliases for readability of the transition table
constexpr auto PD_BOV = pda_state_t::PD_BOV;
constexpr auto PD_BOA = pda_state_t::PD_BOA;
constexpr auto PD_LON = pda_state_t::PD_LON;
constexpr auto PD_STR = pda_state_t::PD_STR;
constexpr auto PD_SCE = pda_state_t::PD_SCE;
constexpr auto PD_PVL = pda_state_t::PD_PVL;
constexpr auto PD_BFN = pda_state_t::PD_BFN;
constexpr auto PD_FLN = pda_state_t::PD_FLN;
constexpr auto PD_FNE = pda_state_t::PD_FNE;
constexpr auto PD_PFN = pda_state_t::PD_PFN;
constexpr auto PD_ERR = pda_state_t::PD_ERR;

constexpr auto PD_NUM_STATES = static_cast<StateT>(pda_state_t::PD_NUM_STATES);

// The starting state of the pushdown automaton
constexpr auto start_state = static_cast<StateT>(pda_state_t::PD_BOV);

/**
 * @brief Getting the transition table
 */
auto get_transition_table(bool newline_delimited_json)
{
  static_assert(static_cast<PdaStackSymbolGroupIdT>(stack_symbol_group_id::STACK_ROOT) == 0);
  static_assert(static_cast<PdaStackSymbolGroupIdT>(stack_symbol_group_id::STACK_LIST) == 1);
  static_assert(static_cast<PdaStackSymbolGroupIdT>(stack_symbol_group_id::STACK_STRUCT) == 2);

  // In case of newline-delimited JSON, multiple newlines are ignored, similar to whitespace.
  // Thas is, empty lines are ignored
  auto const PD_ANL = newline_delimited_json ? PD_BOV : PD_PVL;
  std::array<std::array<pda_state_t, NUM_PDA_SGIDS>, PD_NUM_STATES> pda_tt;
  //  {       [       }       ]       "       \       ,       :     space   newline other
  pda_tt[static_cast<StateT>(pda_state_t::PD_BOV)] = {
    PD_BOA, PD_BOA, PD_ERR, PD_ERR, PD_STR, PD_ERR, PD_ERR, PD_ERR, PD_BOV, PD_BOV, PD_LON,
    PD_BOA, PD_BOA, PD_ERR, PD_ERR, PD_STR, PD_ERR, PD_ERR, PD_ERR, PD_BOV, PD_BOV, PD_LON,
    PD_BOA, PD_BOA, PD_ERR, PD_ERR, PD_STR, PD_ERR, PD_ERR, PD_ERR, PD_BOV, PD_BOV, PD_LON};
  pda_tt[static_cast<StateT>(pda_state_t::PD_BOA)] = {
    PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_ERR,
    PD_BOA, PD_BOA, PD_ERR, PD_PVL, PD_STR, PD_ERR, PD_ERR, PD_ERR, PD_BOA, PD_BOA, PD_LON,
    PD_ERR, PD_ERR, PD_PVL, PD_ERR, PD_FLN, PD_ERR, PD_ERR, PD_ERR, PD_BOA, PD_BOA, PD_ERR};
  pda_tt[static_cast<StateT>(pda_state_t::PD_LON)] = {
    PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_PVL, PD_PVL, PD_LON,
    PD_ERR, PD_ERR, PD_ERR, PD_PVL, PD_ERR, PD_ERR, PD_BOV, PD_ERR, PD_PVL, PD_PVL, PD_LON,
    PD_ERR, PD_ERR, PD_PVL, PD_ERR, PD_ERR, PD_ERR, PD_BFN, PD_ERR, PD_PVL, PD_PVL, PD_LON};
  pda_tt[static_cast<StateT>(pda_state_t::PD_STR)] = {
    PD_STR, PD_STR, PD_STR, PD_STR, PD_PVL, PD_SCE, PD_STR, PD_STR, PD_STR, PD_STR, PD_STR,
    PD_STR, PD_STR, PD_STR, PD_STR, PD_PVL, PD_SCE, PD_STR, PD_STR, PD_STR, PD_STR, PD_STR,
    PD_STR, PD_STR, PD_STR, PD_STR, PD_PVL, PD_SCE, PD_STR, PD_STR, PD_STR, PD_STR, PD_STR};
  pda_tt[static_cast<StateT>(pda_state_t::PD_SCE)] = {
    PD_STR, PD_STR, PD_STR, PD_STR, PD_STR, PD_STR, PD_STR, PD_STR, PD_STR, PD_STR, PD_STR,
    PD_STR, PD_STR, PD_STR, PD_STR, PD_STR, PD_STR, PD_STR, PD_STR, PD_STR, PD_STR, PD_STR,
    PD_STR, PD_STR, PD_STR, PD_STR, PD_STR, PD_STR, PD_STR, PD_STR, PD_STR, PD_STR, PD_STR};
  pda_tt[static_cast<StateT>(pda_state_t::PD_PVL)] = {
    PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_PVL, PD_ANL, PD_ERR,
    PD_ERR, PD_ERR, PD_ERR, PD_PVL, PD_ERR, PD_ERR, PD_BOV, PD_ERR, PD_PVL, PD_PVL, PD_ERR,
    PD_ERR, PD_ERR, PD_PVL, PD_ERR, PD_ERR, PD_ERR, PD_BFN, PD_ERR, PD_PVL, PD_PVL, PD_ERR};
  pda_tt[static_cast<StateT>(pda_state_t::PD_BFN)] = {
    PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_ERR,
    PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_ERR,
    PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_FLN, PD_ERR, PD_ERR, PD_ERR, PD_BFN, PD_BFN, PD_ERR};
  pda_tt[static_cast<StateT>(pda_state_t::PD_FLN)] = {
    PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_ERR,
    PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_ERR,
    PD_FLN, PD_FLN, PD_FLN, PD_FLN, PD_PFN, PD_FNE, PD_FLN, PD_FLN, PD_FLN, PD_FLN, PD_FLN};
  pda_tt[static_cast<StateT>(pda_state_t::PD_FNE)] = {
    PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_ERR,
    PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_ERR,
    PD_FLN, PD_FLN, PD_FLN, PD_FLN, PD_FLN, PD_FLN, PD_FLN, PD_FLN, PD_FLN, PD_FLN, PD_FLN};
  pda_tt[static_cast<StateT>(pda_state_t::PD_PFN)] = {
    PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_ERR,
    PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_ERR,
    PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_BOV, PD_PFN, PD_PFN, PD_ERR};
  pda_tt[static_cast<StateT>(pda_state_t::PD_ERR)] = {
    PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_ERR,
    PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_ERR,
    PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_ERR, PD_ERR};
  return pda_tt;
}

/**
 * @brief Getting the translation table
 */
auto get_translation_table()
{
  constexpr auto StructBegin       = token_t::StructBegin;
  constexpr auto StructEnd         = token_t::StructEnd;
  constexpr auto ListBegin         = token_t::ListBegin;
  constexpr auto ListEnd           = token_t::ListEnd;
  constexpr auto StructMemberBegin = token_t::StructMemberBegin;
  constexpr auto StructMemberEnd   = token_t::StructMemberEnd;
  constexpr auto FieldNameBegin    = token_t::FieldNameBegin;
  constexpr auto FieldNameEnd      = token_t::FieldNameEnd;
  constexpr auto StringBegin       = token_t::StringBegin;
  constexpr auto StringEnd         = token_t::StringEnd;
  constexpr auto ValueBegin        = token_t::ValueBegin;
  constexpr auto ValueEnd          = token_t::ValueEnd;
  constexpr auto ErrorBegin        = token_t::ErrorBegin;

  std::array<std::array<std::vector<char>, NUM_PDA_SGIDS>, PD_NUM_STATES> pda_tlt;
  pda_tlt[static_cast<StateT>(pda_state_t::PD_BOV)] = {{                /*ROOT*/
                                                        {StructBegin},  // OPENING_BRACE
                                                        {ListBegin},    // OPENING_BRACKET
                                                        {ErrorBegin},   // CLOSING_BRACE
                                                        {ErrorBegin},   // CLOSING_BRACKET
                                                        {StringBegin},  // QUOTE
                                                        {ErrorBegin},   // ESCAPE
                                                        {ErrorBegin},   // COMMA
                                                        {ErrorBegin},   // COLON
                                                        {},             // WHITE_SPACE
                                                        {},             // LINE_BREAK
                                                        {ValueBegin},   // OTHER
                                                        /*LIST*/
                                                        {StructBegin},  // OPENING_BRACE
                                                        {ListBegin},    // OPENING_BRACKET
                                                        {ErrorBegin},   // CLOSING_BRACE
                                                        {ErrorBegin},   // CLOSING_BRACKET
                                                        {StringBegin},  // QUOTE
                                                        {ErrorBegin},   // ESCAPE
                                                        {ErrorBegin},   // COMMA
                                                        {ErrorBegin},   // COLON
                                                        {},             // WHITE_SPACE
                                                        {},             // LINE_BREAK
                                                        {ValueBegin},   // OTHER
                                                        /*STRUCT*/
                                                        {StructBegin},   // OPENING_BRACE
                                                        {ListBegin},     // OPENING_BRACKET
                                                        {ErrorBegin},    // CLOSING_BRACE
                                                        {ErrorBegin},    // CLOSING_BRACKET
                                                        {StringBegin},   // QUOTE
                                                        {ErrorBegin},    // ESCAPE
                                                        {ErrorBegin},    // COMMA
                                                        {ErrorBegin},    // COLON
                                                        {},              // WHITE_SPACE
                                                        {},              // LINE_BREAK
                                                        {ValueBegin}}};  // OTHER
  pda_tlt[static_cast<StateT>(pda_state_t::PD_BOA)] = {
    {               /*ROOT*/
     {ErrorBegin},  // OPENING_BRACE
     {ErrorBegin},  // OPENING_BRACKET
     {ErrorBegin},  // CLOSING_BRACE
     {ErrorBegin},  // CLOSING_BRACKET
     {ErrorBegin},  // QUOTE
     {ErrorBegin},  // ESCAPE
     {ErrorBegin},  // COMMA
     {ErrorBegin},  // COLON
     {ErrorBegin},  // WHITE_SPACE
     {ErrorBegin},  // LINE_BREAK
     {ErrorBegin},  // OTHER
     /*LIST*/
     {StructBegin},  // OPENING_BRACE
     {ListBegin},    // OPENING_BRACKET
     {ErrorBegin},   // CLOSING_BRACE
     {ListEnd},      // CLOSING_BRACKET
     {StringBegin},  // QUOTE
     {ErrorBegin},   // ESCAPE
     {ErrorBegin},   // COMMA
     {ErrorBegin},   // COLON
     {},             // WHITE_SPACE
     {},             // LINE_BREAK
     {ValueBegin},   // OTHER
     /*STRUCT*/
     {ErrorBegin},                         // OPENING_BRACE
     {ErrorBegin},                         // OPENING_BRACKET
     {StructEnd},                          // CLOSING_BRACE
     {ErrorBegin},                         // CLOSING_BRACKET
     {StructMemberBegin, FieldNameBegin},  // QUOTE
     {ErrorBegin},                         // ESCAPE
     {ErrorBegin},                         // COMMA
     {ErrorBegin},                         // COLON
     {},                                   // WHITE_SPACE
     {},                                   // LINE_BREAK
     {ErrorBegin}}};                       // OTHER
  pda_tlt[static_cast<StateT>(pda_state_t::PD_LON)] = {
    {               /*ROOT*/
     {ErrorBegin},  // OPENING_BRACE
     {ErrorBegin},  // OPENING_BRACKET
     {ErrorBegin},  // CLOSING_BRACE
     {ErrorBegin},  // CLOSING_BRACKET
     {ErrorBegin},  // QUOTE
     {ErrorBegin},  // ESCAPE
     {ErrorBegin},  // COMMA
     {ErrorBegin},  // COLON
     {ValueEnd},    // WHITE_SPACE
     {ValueEnd},    // LINE_BREAK
     {},            // OTHER
     /*LIST*/
     {ErrorBegin},         // OPENING_BRACE
     {ErrorBegin},         // OPENING_BRACKET
     {ErrorBegin},         // CLOSING_BRACE
     {ValueEnd, ListEnd},  // CLOSING_BRACKET
     {ErrorBegin},         // QUOTE
     {ErrorBegin},         // ESCAPE
     {ValueEnd},           // COMMA
     {ErrorBegin},         // COLON
     {ValueEnd},           // WHITE_SPACE
     {ValueEnd},           // LINE_BREAK
     {},                   // OTHER
     /*STRUCT*/
     {ErrorBegin},                            // OPENING_BRACE
     {ErrorBegin},                            // OPENING_BRACKET
     {ValueEnd, StructMemberEnd, StructEnd},  // CLOSING_BRACE
     {ErrorBegin},                            // CLOSING_BRACKET
     {ErrorBegin},                            // QUOTE
     {ErrorBegin},                            // ESCAPE
     {ValueEnd, StructMemberEnd},             // COMMA
     {ErrorBegin},                            // COLON
     {ValueEnd},                              // WHITE_SPACE
     {ValueEnd},                              // LINE_BREAK
     {}}};                                    // OTHER

  pda_tlt[static_cast<StateT>(pda_state_t::PD_STR)] = {{              /*ROOT*/
                                                        {},           // OPENING_BRACE
                                                        {},           // OPENING_BRACKET
                                                        {},           // CLOSING_BRACE
                                                        {},           // CLOSING_BRACKET
                                                        {StringEnd},  // QUOTE
                                                        {},           // ESCAPE
                                                        {},           // COMMA
                                                        {},           // COLON
                                                        {},           // WHITE_SPACE
                                                        {},           // LINE_BREAK
                                                        {},           // OTHER
                                                        /*LIST*/
                                                        {},           // OPENING_BRACE
                                                        {},           // OPENING_BRACKET
                                                        {},           // CLOSING_BRACE
                                                        {},           // CLOSING_BRACKET
                                                        {StringEnd},  // QUOTE
                                                        {},           // ESCAPE
                                                        {},           // COMMA
                                                        {},           // COLON
                                                        {},           // WHITE_SPACE
                                                        {},           // LINE_BREAK
                                                        {},           // OTHER
                                                        /*STRUCT*/
                                                        {},           // OPENING_BRACE
                                                        {},           // OPENING_BRACKET
                                                        {},           // CLOSING_BRACE
                                                        {},           // CLOSING_BRACKET
                                                        {StringEnd},  // QUOTE
                                                        {},           // ESCAPE
                                                        {},           // COMMA
                                                        {},           // COLON
                                                        {},           // WHITE_SPACE
                                                        {},           // LINE_BREAK
                                                        {}}};         // OTHER

  pda_tlt[static_cast<StateT>(pda_state_t::PD_SCE)] = {{     /*ROOT*/
                                                        {},  // OPENING_BRACE
                                                        {},  // OPENING_BRACKET
                                                        {},  // CLOSING_BRACE
                                                        {},  // CLOSING_BRACKET
                                                        {},  // QUOTE
                                                        {},  // ESCAPE
                                                        {},  // COMMA
                                                        {},  // COLON
                                                        {},  // WHITE_SPACE
                                                        {},  // LINE_BREAK
                                                        {},  // OTHER
                                                        /*LIST*/
                                                        {},  // OPENING_BRACE
                                                        {},  // OPENING_BRACKET
                                                        {},  // CLOSING_BRACE
                                                        {},  // CLOSING_BRACKET
                                                        {},  // QUOTE
                                                        {},  // ESCAPE
                                                        {},  // COMMA
                                                        {},  // COLON
                                                        {},  // WHITE_SPACE
                                                        {},  // LINE_BREAK
                                                        {},  // OTHER
                                                        /*STRUCT*/
                                                        {},    // OPENING_BRACE
                                                        {},    // OPENING_BRACKET
                                                        {},    // CLOSING_BRACE
                                                        {},    // CLOSING_BRACKET
                                                        {},    // QUOTE
                                                        {},    // ESCAPE
                                                        {},    // COMMA
                                                        {},    // COLON
                                                        {},    // WHITE_SPACE
                                                        {},    // LINE_BREAK
                                                        {}}};  // OTHER

  pda_tlt[static_cast<StateT>(pda_state_t::PD_PVL)] = {
    {               /*ROOT*/
     {ErrorBegin},  // OPENING_BRACE
     {ErrorBegin},  // OPENING_BRACKET
     {ErrorBegin},  // CLOSING_BRACE
     {ErrorBegin},  // CLOSING_BRACKET
     {ErrorBegin},  // QUOTE
     {ErrorBegin},  // ESCAPE
     {ErrorBegin},  // COMMA
     {ErrorBegin},  // COLON
     {},            // WHITE_SPACE
     {},            // LINE_BREAK
     {ErrorBegin},  // OTHER
     /*LIST*/
     {ErrorBegin},  // OPENING_BRACE
     {ErrorBegin},  // OPENING_BRACKET
     {ErrorBegin},  // CLOSING_BRACE
     {ListEnd},     // CLOSING_BRACKET
     {ErrorBegin},  // QUOTE
     {ErrorBegin},  // ESCAPE
     {},            // COMMA
     {ErrorBegin},  // COLON
     {},            // WHITE_SPACE
     {},            // LINE_BREAK
     {ErrorBegin},  // OTHER
     /*STRUCT*/
     {ErrorBegin},                  // OPENING_BRACE
     {ErrorBegin},                  // OPENING_BRACKET
     {StructMemberEnd, StructEnd},  // CLOSING_BRACE
     {ErrorBegin},                  // CLOSING_BRACKET
     {ErrorBegin},                  // QUOTE
     {ErrorBegin},                  // ESCAPE
     {StructMemberEnd},             // COMMA
     {ErrorBegin},                  // COLON
     {},                            // WHITE_SPACE
     {},                            // LINE_BREAK
     {ErrorBegin}}};                // OTHER

  pda_tlt[static_cast<StateT>(pda_state_t::PD_BFN)] = {
    {               /*ROOT*/
     {ErrorBegin},  // OPENING_BRACE
     {ErrorBegin},  // OPENING_BRACKET
     {ErrorBegin},  // CLOSING_BRACE
     {ErrorBegin},  // CLOSING_BRACKET
     {ErrorBegin},  // QUOTE
     {ErrorBegin},  // ESCAPE
     {ErrorBegin},  // COMMA
     {ErrorBegin},  // COLON
     {ErrorBegin},  // WHITE_SPACE
     {ErrorBegin},  // LINE_BREAK
     {ErrorBegin},  // OTHER
     /*LIST*/
     {ErrorBegin},  // OPENING_BRACE
     {ErrorBegin},  // OPENING_BRACKET
     {ErrorBegin},  // CLOSING_BRACE
     {ErrorBegin},  // CLOSING_BRACKET
     {ErrorBegin},  // QUOTE
     {ErrorBegin},  // ESCAPE
     {ErrorBegin},  // COMMA
     {ErrorBegin},  // COLON
     {ErrorBegin},  // WHITE_SPACE
     {ErrorBegin},  // LINE_BREAK
     {ErrorBegin},  // OTHER
     /*STRUCT*/
     {ErrorBegin},                         // OPENING_BRACE
     {ErrorBegin},                         // OPENING_BRACKET
     {ErrorBegin},                         // CLOSING_BRACE
     {ErrorBegin},                         // CLOSING_BRACKET
     {StructMemberBegin, FieldNameBegin},  // QUOTE
     {ErrorBegin},                         // ESCAPE
     {ErrorBegin},                         // COMMA
     {ErrorBegin},                         // COLON
     {},                                   // WHITE_SPACE
     {},                                   // LINE_BREAK
     {ErrorBegin}}};                       // OTHER

  pda_tlt[static_cast<StateT>(pda_state_t::PD_FLN)] = {{               /*ROOT*/
                                                        {ErrorBegin},  // OPENING_BRACE
                                                        {ErrorBegin},  // OPENING_BRACKET
                                                        {ErrorBegin},  // CLOSING_BRACE
                                                        {ErrorBegin},  // CLOSING_BRACKET
                                                        {ErrorBegin},  // QUOTE
                                                        {ErrorBegin},  // ESCAPE
                                                        {ErrorBegin},  // COMMA
                                                        {ErrorBegin},  // COLON
                                                        {ErrorBegin},  // WHITE_SPACE
                                                        {ErrorBegin},  // LINE_BREAK
                                                        {ErrorBegin},  // OTHER
                                                        /*LIST*/
                                                        {ErrorBegin},  // OPENING_BRACE
                                                        {ErrorBegin},  // OPENING_BRACKET
                                                        {ErrorBegin},  // CLOSING_BRACE
                                                        {ErrorBegin},  // CLOSING_BRACKET
                                                        {ErrorBegin},  // QUOTE
                                                        {ErrorBegin},  // ESCAPE
                                                        {ErrorBegin},  // COMMA
                                                        {ErrorBegin},  // COLON
                                                        {ErrorBegin},  // WHITE_SPACE
                                                        {ErrorBegin},  // LINE_BREAK
                                                        {ErrorBegin},  // OTHER
                                                        /*STRUCT*/
                                                        {},              // OPENING_BRACE
                                                        {},              // OPENING_BRACKET
                                                        {},              // CLOSING_BRACE
                                                        {},              // CLOSING_BRACKET
                                                        {FieldNameEnd},  // QUOTE
                                                        {},              // ESCAPE
                                                        {},              // COMMA
                                                        {},              // COLON
                                                        {},              // WHITE_SPACE
                                                        {},              // LINE_BREAK
                                                        {}}};            // OTHER

  pda_tlt[static_cast<StateT>(pda_state_t::PD_FNE)] = {{               /*ROOT*/
                                                        {ErrorBegin},  // OPENING_BRACE
                                                        {ErrorBegin},  // OPENING_BRACKET
                                                        {ErrorBegin},  // CLOSING_BRACE
                                                        {ErrorBegin},  // CLOSING_BRACKET
                                                        {ErrorBegin},  // QUOTE
                                                        {ErrorBegin},  // ESCAPE
                                                        {ErrorBegin},  // COMMA
                                                        {ErrorBegin},  // COLON
                                                        {ErrorBegin},  // WHITE_SPACE
                                                        {ErrorBegin},  // LINE_BREAK
                                                        {ErrorBegin},  // OTHER
                                                        /*LIST*/
                                                        {ErrorBegin},  // OPENING_BRACE
                                                        {ErrorBegin},  // OPENING_BRACKET
                                                        {ErrorBegin},  // CLOSING_BRACE
                                                        {ErrorBegin},  // CLOSING_BRACKET
                                                        {ErrorBegin},  // QUOTE
                                                        {ErrorBegin},  // ESCAPE
                                                        {ErrorBegin},  // COMMA
                                                        {ErrorBegin},  // COLON
                                                        {ErrorBegin},  // WHITE_SPACE
                                                        {ErrorBegin},  // LINE_BREAK
                                                        {ErrorBegin},  // OTHER
                                                        /*STRUCT*/
                                                        {},    // OPENING_BRACE
                                                        {},    // OPENING_BRACKET
                                                        {},    // CLOSING_BRACE
                                                        {},    // CLOSING_BRACKET
                                                        {},    // QUOTE
                                                        {},    // ESCAPE
                                                        {},    // COMMA
                                                        {},    // COLON
                                                        {},    // WHITE_SPACE
                                                        {},    // LINE_BREAK
                                                        {}}};  // OTHER

  pda_tlt[static_cast<StateT>(pda_state_t::PD_PFN)] = {{               /*ROOT*/
                                                        {ErrorBegin},  // OPENING_BRACE
                                                        {ErrorBegin},  // OPENING_BRACKET
                                                        {ErrorBegin},  // CLOSING_BRACE
                                                        {ErrorBegin},  // CLOSING_BRACKET
                                                        {ErrorBegin},  // QUOTE
                                                        {ErrorBegin},  // ESCAPE
                                                        {ErrorBegin},  // COMMA
                                                        {ErrorBegin},  // COLON
                                                        {ErrorBegin},  // WHITE_SPACE
                                                        {ErrorBegin},  // LINE_BREAK
                                                        {ErrorBegin},  // OTHER
                                                        /*LIST*/
                                                        {ErrorBegin},  // OPENING_BRACE
                                                        {ErrorBegin},  // OPENING_BRACKET
                                                        {ErrorBegin},  // CLOSING_BRACE
                                                        {ErrorBegin},  // CLOSING_BRACKET
                                                        {ErrorBegin},  // QUOTE
                                                        {ErrorBegin},  // ESCAPE
                                                        {ErrorBegin},  // COMMA
                                                        {ErrorBegin},  // COLON
                                                        {ErrorBegin},  // WHITE_SPACE
                                                        {ErrorBegin},  // LINE_BREAK
                                                        {ErrorBegin},  // OTHER
                                                        /*STRUCT*/
                                                        {ErrorBegin},    // OPENING_BRACE
                                                        {ErrorBegin},    // OPENING_BRACKET
                                                        {ErrorBegin},    // CLOSING_BRACE
                                                        {ErrorBegin},    // CLOSING_BRACKET
                                                        {ErrorBegin},    // QUOTE
                                                        {ErrorBegin},    // ESCAPE
                                                        {ErrorBegin},    // COMMA
                                                        {},              // COLON
                                                        {},              // WHITE_SPACE
                                                        {},              // LINE_BREAK
                                                        {ErrorBegin}}};  // OTHER

  pda_tlt[static_cast<StateT>(pda_state_t::PD_ERR)] = {{     /*ROOT*/
                                                        {},  // OPENING_BRACE
                                                        {},  // OPENING_BRACKET
                                                        {},  // CLOSING_BRACE
                                                        {},  // CLOSING_BRACKET
                                                        {},  // QUOTE
                                                        {},  // ESCAPE
                                                        {},  // COMMA
                                                        {},  // COLON
                                                        {},  // WHITE_SPACE
                                                        {},  // LINE_BREAK
                                                        {},  // OTHER
                                                        /*LIST*/
                                                        {},  // OPENING_BRACE
                                                        {},  // OPENING_BRACKET
                                                        {},  // CLOSING_BRACE
                                                        {},  // CLOSING_BRACKET
                                                        {},  // QUOTE
                                                        {},  // ESCAPE
                                                        {},  // COMMA
                                                        {},  // COLON
                                                        {},  // WHITE_SPACE
                                                        {},  // LINE_BREAK
                                                        {},  // OTHER
                                                        /*STRUCT*/
                                                        {},    // OPENING_BRACE
                                                        {},    // OPENING_BRACKET
                                                        {},    // CLOSING_BRACE
                                                        {},    // CLOSING_BRACKET
                                                        {},    // QUOTE
                                                        {},    // ESCAPE
                                                        {},    // COMMA
                                                        {},    // COLON
                                                        {},    // WHITE_SPACE
                                                        {},    // LINE_BREAK
                                                        {}}};  // OTHER
  return pda_tlt;
}

}  // namespace tokenizer_pda

/**
 * @brief Function object used to filter for brackets and braces that represent push and pop
 * operations
 */
struct JSONToStackOp {
  template <typename StackSymbolT>
  constexpr CUDF_HOST_DEVICE fst::stack_op_type operator()(StackSymbolT const& stack_symbol) const
  {
    return (stack_symbol == '{' || stack_symbol == '[')   ? fst::stack_op_type::PUSH
           : (stack_symbol == '}' || stack_symbol == ']') ? fst::stack_op_type::POP
                                                          : fst::stack_op_type::READ;
  }
};

void json_column::null_fill(row_offset_t up_to_row_offset)
{
  // Fill all the rows up to up_to_row_offset with "empty"/null rows
  validity.resize(word_index(up_to_row_offset) + 1);
  std::fill_n(std::back_inserter(string_offsets),
              up_to_row_offset - string_offsets.size(),
              (string_offsets.size() > 0) ? string_offsets.back() : 0);
  std::fill_n(std::back_inserter(string_lengths), up_to_row_offset - string_lengths.size(), 0);
  std::fill_n(std::back_inserter(child_offsets),
              up_to_row_offset + 1 - child_offsets.size(),
              (child_offsets.size() > 0) ? child_offsets.back() : 0);
  current_offset = up_to_row_offset;
}

void json_column::level_child_cols_recursively(row_offset_t min_row_count)
{
  // Fill this columns with nulls up to the given row count
  null_fill(min_row_count);

  // If this is a struct column, we need to level all its child columns
  if (type == json_col_t::StructColumn) {
    for (auto it = std::begin(child_columns); it != std::end(child_columns); it++) {
      it->second.level_child_cols_recursively(min_row_count);
    }
  }
  // If this is a list column, we need to make sure that its child column levels its children
  else if (type == json_col_t::ListColumn) {
    auto it = std::begin(child_columns);
    // Make that child column fill its child columns up to its own row count
    if (it != std::end(child_columns)) {
      it->second.level_child_cols_recursively(it->second.current_offset);
    }
  }
};

void json_column::append_row(uint32_t row_index,
                             json_col_t row_type,
                             uint32_t string_offset,
                             uint32_t string_end,
                             uint32_t child_count)
{
  // If, thus far, the column's type couldn't be inferred, we infer it to the given type
  if (type == json_col_t::Unknown) {
    type = row_type;
  }
  // If, at some point within a column, we encounter a nested type (list or struct),
  // we change that column's type to that respective nested type and invalidate all previous rows
  else if (type == json_col_t::StringColumn &&
           (row_type == json_col_t::ListColumn || row_type == json_col_t::StructColumn)) {
    // Change the column type
    type = row_type;

    // Invalidate all previous entries, as they were _not_ of the nested type to which we just
    // converted
    std::fill_n(validity.begin(), validity.size(), 0);
    valid_count = 0U;
  }
  // If this is a nested column but we're trying to insert either (a) a list node into a struct
  // column or (b) a struct node into a list column, we fail
  CUDF_EXPECTS(not((type == json_col_t::ListColumn and row_type == json_col_t::StructColumn) or
                   (type == json_col_t::StructColumn and row_type == json_col_t::ListColumn)),
               "A mix of lists and structs within the same column is not supported");

  // We shouldn't run into this, as we shouldn't be asked to append an "unknown" row type
  CUDF_EXPECTS(type != json_col_t::Unknown, "Encountered invalid JSON token sequence");

  // Fill all the omitted rows with "empty"/null rows (if needed)
  null_fill(row_index);

  // Table listing what we intend to use for a given column type and row type combination
  // col type | row type  => {valid, FAIL, null}
  // -----------------------------------------------
  // List     | List      => valid
  // List     | Struct    => FAIL
  // List     | String    => null
  // Struct   | List      => FAIL
  // Struct   | Struct    => valid
  // Struct   | String    => null
  // String   | List      => valid (we switch col type to list, null'ing all previous rows)
  // String   | Struct    => valid (we switch col type to list, null'ing all previous rows)
  // String   | String    => valid
  bool const is_valid = (type == row_type);
  if (static_cast<size_type>(validity.size()) < word_index(current_offset)) validity.push_back({});
  if (is_valid) { set_bit_unsafe(&validity.back(), intra_word_index(current_offset)); }
  valid_count += (is_valid) ? 1U : 0U;
  string_offsets.push_back(string_offset);
  string_lengths.push_back(string_end - string_offset);
  child_offsets.push_back((child_offsets.size() > 0) ? child_offsets.back() + child_count : 0);
  current_offset++;
};

namespace detail {

void get_stack_context(device_span<SymbolT const> json_in,
                       SymbolT* d_top_of_stack,
                       rmm::cuda_stream_view stream)
{
  // Range of encapsulating function that comprises:
  // -> DFA simulation for filtering out brackets and braces inside of quotes
  // -> Logical stack to infer the stack context
  CUDF_FUNC_RANGE();

  // Symbol representing the JSON-root (i.e., we're at nesting level '0')
  constexpr StackSymbolT root_symbol = '_';
  // This can be any stack symbol from the stack alphabet that does not push onto stack
  constexpr StackSymbolT read_symbol = 'x';

  // Number of stack operations in the input (i.e., number of '{', '}', '[', ']' outside of quotes)
  rmm::device_scalar<SymbolOffsetT> d_num_stack_ops(stream);

  // Sequence of stack symbols and their position in the original input (sparse representation)
  rmm::device_uvector<StackSymbolT> stack_ops{json_in.size(), stream};
  rmm::device_uvector<SymbolOffsetT> stack_op_indices{json_in.size(), stream};

  // Prepare finite-state transducer that only selects '{', '}', '[', ']' outside of quotes
  using ToStackOpFstT =
    cudf::io::fst::detail::Dfa<StackSymbolT,
                               static_cast<int32_t>(
                                 to_stack_op::dfa_symbol_group_id::NUM_SYMBOL_GROUPS),
                               static_cast<int32_t>(to_stack_op::dfa_states::TT_NUM_STATES)>;
  ToStackOpFstT json_to_stack_ops_fst{to_stack_op::symbol_groups,
                                      to_stack_op::transition_table,
                                      to_stack_op::translation_table,
                                      stream};

  // "Search" for relevant occurrence of brackets and braces that indicate the beginning/end
  // of structs/lists
  json_to_stack_ops_fst.Transduce(json_in.begin(),
                                  static_cast<SymbolOffsetT>(json_in.size()),
                                  stack_ops.data(),
                                  stack_op_indices.data(),
                                  d_num_stack_ops.data(),
                                  to_stack_op::start_state,
                                  stream);

  // Copy back to actual number of stack operations
  auto const num_stack_ops = d_num_stack_ops.value(stream);

  // stack operations with indices are converted to top of the stack for each character in the input
  fst::sparse_stack_op_to_top_of_stack<StackLevelT>(
    stack_ops.data(),
    device_span<SymbolOffsetT>{stack_op_indices.data(), num_stack_ops},
    JSONToStackOp{},
    d_top_of_stack,
    root_symbol,
    read_symbol,
    json_in.size(),
    stream);
}

std::pair<rmm::device_uvector<PdaTokenT>, rmm::device_uvector<SymbolOffsetT>> get_token_stream(
  device_span<SymbolT const> json_in,
  cudf::io::json_reader_options const& options,
  rmm::cuda_stream_view stream,
  rmm::mr::device_memory_resource* mr)
{
  // Range of encapsulating function that parses to internal columnar data representation
  CUDF_FUNC_RANGE();

  auto const new_line_delimited_json = options.is_enabled_lines();

  // Prepare for PDA transducer pass, merging input symbols with stack symbols
  rmm::device_uvector<PdaSymbolGroupIdT> pda_sgids = [json_in, stream]() {
    rmm::device_uvector<PdaSymbolGroupIdT> pda_sgids{json_in.size(), stream};
    // Memory holding the top-of-stack stack context for the input
    rmm::device_uvector<StackSymbolT> stack_op_indices{json_in.size(), stream};

    // Identify what is the stack context for each input character (JSON-root, struct, or list)
    get_stack_context(json_in, stack_op_indices.data(), stream);

    auto zip_in = thrust::make_zip_iterator(json_in.data(), stack_op_indices.data());
    thrust::transform(rmm::exec_policy(stream),
                      zip_in,
                      zip_in + json_in.size(),
                      pda_sgids.data(),
                      tokenizer_pda::PdaSymbolToSymbolGroupId{});
    return pda_sgids;
  }();

  // PDA transducer alias
  using ToTokenStreamFstT =
    cudf::io::fst::detail::Dfa<StackSymbolT,
                               tokenizer_pda::NUM_PDA_SGIDS,
                               static_cast<tokenizer_pda::StateT>(
                                 tokenizer_pda::pda_state_t::PD_NUM_STATES)>;

  // Instantiating PDA transducer
  std::vector<std::vector<char>> pda_sgid_identity{tokenizer_pda::NUM_PDA_SGIDS};
  std::generate(std::begin(pda_sgid_identity),
                std::end(pda_sgid_identity),
                [i = char{0}]() mutable { return std::vector<char>{i++}; });
  ToTokenStreamFstT json_to_tokens_fst{pda_sgid_identity,
                                       tokenizer_pda::get_transition_table(new_line_delimited_json),
                                       tokenizer_pda::get_translation_table(),
                                       stream};

  // Perform a PDA-transducer pass
  // Compute the maximum amount of tokens that can possibly be emitted for a given input size
  // Worst case ratio of tokens per input char is given for a struct with an empty field name, that
  // may be arbitrarily deeply nested: {"":_}, where '_' is a placeholder for any JSON value,
  // possibly another such struct. That is, 6 tokens for 5 chars (plus chars and tokens of '_')
  std::size_t constexpr min_chars_per_struct  = 5;
  std::size_t constexpr max_tokens_per_struct = 6;
  auto const max_token_out_count =
    cudf::util::div_rounding_up_safe(json_in.size(), min_chars_per_struct) * max_tokens_per_struct;
  rmm::device_scalar<SymbolOffsetT> num_written_tokens{stream};
  rmm::device_uvector<PdaTokenT> tokens{max_token_out_count, stream, mr};
  rmm::device_uvector<SymbolOffsetT> tokens_indices{max_token_out_count, stream, mr};

  json_to_tokens_fst.Transduce(pda_sgids.begin(),
                               static_cast<SymbolOffsetT>(json_in.size()),
                               tokens.data(),
                               tokens_indices.data(),
                               num_written_tokens.data(),
                               tokenizer_pda::start_state,
                               stream);

  auto const num_total_tokens = num_written_tokens.value(stream);
  tokens.resize(num_total_tokens, stream);
  tokens_indices.resize(num_total_tokens, stream);

  CUDF_EXPECTS(num_total_tokens <= max_token_out_count,
               "Generated token count exceeds the expected token count");

  return std::make_pair(std::move(tokens), std::move(tokens_indices));
}

/**
 * @brief Parses the given JSON string and generates a tree representation of the given input.
 *
 * @param[in,out] root_column The root column of the hierarchy of columns into which data is parsed
 * @param[in,out] current_data_path The stack represents the path from the JSON root node to the
 * first node encountered in \p input
 * @param[in] input The JSON input in host memory
 * @param[in] d_input The JSON input in device memory
 * @param[in] options Parsing options specifying the parsing behaviour
 * @param[in] include_quote_char Whether to include the original quote chars around string values,
 * allowing to distinguish string values from numeric and literal values
 * @param[in] stream The CUDA stream to which kernels are dispatched
 * @param[in] mr Optional, resource with which to allocate
 * @return The columnar representation of the data from the given JSON input
 */
void make_json_column(json_column& root_column,
                      std::stack<tree_node>& current_data_path,
                      host_span<SymbolT const> input,
                      device_span<SymbolT const> d_input,
                      cudf::io::json_reader_options const& options,
                      bool include_quote_char,
                      rmm::cuda_stream_view stream,
                      rmm::mr::device_memory_resource* mr = rmm::mr::get_current_device_resource())
{
  // Range of encapsulating function that parses to internal columnar data representation
  CUDF_FUNC_RANGE();

  // Parse the JSON and get the token stream
  const auto [d_tokens_gpu, d_token_indices_gpu] = get_token_stream(d_input, options, stream, mr);

  // Copy the JSON tokens to the host
  thrust::host_vector<PdaTokenT> tokens =
    cudf::detail::make_host_vector_async(d_tokens_gpu, stream);
  thrust::host_vector<SymbolOffsetT> token_indices_gpu =
    cudf::detail::make_host_vector_async(d_token_indices_gpu, stream);

  // Make sure tokens have been copied to the host
  stream.synchronize();

  // Whether this token is the valid token to begin the JSON document with
  auto is_valid_root_token = [](PdaTokenT const token) {
    switch (token) {
      case token_t::StructBegin:
      case token_t::ListBegin:
      case token_t::StringBegin:
      case token_t::ValueBegin: return true;
      default: return false;
    };
  };

  // Returns the token's corresponding column type
  auto token_to_column_type = [](PdaTokenT const token) {
    switch (token) {
      case token_t::StructBegin: return json_col_t::StructColumn;
      case token_t::ListBegin: return json_col_t::ListColumn;
      case token_t::StringBegin: return json_col_t::StringColumn;
      case token_t::ValueBegin: return json_col_t::StringColumn;
      default: return json_col_t::Unknown;
    };
  };

  // Depending on whether we want to include the quotes of strings or not, respectively, we:
  // (a) strip off the beginning quote included in StringBegin and FieldNameBegin or
  // (b) include of the end quote excluded from in StringEnd and strip off the beginning quote
  // included FieldNameBegin
  auto get_token_index = [include_quote_char](PdaTokenT const token,
                                              SymbolOffsetT const token_index) {
    constexpr SymbolOffsetT quote_char_size = 1;
    switch (token) {
      // Optionally strip off quote char included for StringBegin
      case token_t::StringBegin: return token_index + (include_quote_char ? 0 : quote_char_size);
      // Optionally include trailing quote char for string values excluded for StringEnd
      case token_t::StringEnd: return token_index + (include_quote_char ? quote_char_size : 0);
      // Strip off quote char included for FieldNameBegin
      case token_t::FieldNameBegin: return token_index + quote_char_size;
      default: return token_index;
    };
  };

  // The end-of-* partner token for a given beginning-of-* token
  auto end_of_partner = [](PdaTokenT const token) {
    switch (token) {
      case token_t::StringBegin: return token_t::StringEnd;
      case token_t::ValueBegin: return token_t::ValueEnd;
      case token_t::FieldNameBegin: return token_t::FieldNameEnd;
      default: return token_t::ErrorBegin;
    };
  };

#ifdef NJP_DEBUG_PRINT
  auto column_type_string = [](json_col_t column_type) {
    switch (column_type) {
      case json_col_t::Unknown: return "Unknown";
      case json_col_t::ListColumn: return "List";
      case json_col_t::StructColumn: return "Struct";
      case json_col_t::StringColumn: return "String";
      default: return "Unknown";
    }
  };

  auto token_to_string = [](PdaTokenT token_type) {
    switch (token_type) {
      case token_t::StructBegin: return "StructBegin";
      case token_t::StructEnd: return "StructEnd";
      case token_t::ListBegin: return "ListBegin";
      case token_t::ListEnd: return "ListEnd";
      case token_t::StructMemberBegin: return "StructMemberBegin";
      case token_t::StructMemberEnd: return "StructMemberEnd";
      case token_t::FieldNameBegin: return "FieldNameBegin";
      case token_t::FieldNameEnd: return "FieldNameEnd";
      case token_t::StringBegin: return "StringBegin";
      case token_t::StringEnd: return "StringEnd";
      case token_t::ValueBegin: return "ValueBegin";
      case token_t::ValueEnd: return "ValueEnd";
      case token_t::ErrorBegin: return "ErrorBegin";
      default: return "Unknown";
    }
  };
#endif

  /**
   * @brief Updates the given row in the given column with a new string_end and child_count. In
   * particular, updating the child count is relevant for list columns.
   */
  auto update_row =
    [](json_column* column, uint32_t row_index, uint32_t string_end, uint32_t child_count) {
#ifdef NJP_DEBUG_PRINT
      std::cout << "  -> update_row()\n";
      std::cout << "  ---> col@" << column << "\n";
      std::cout << "  ---> row #" << row_index << "\n";
      std::cout << "  ---> string_lengths = " << (string_end - column->string_offsets[row_index])
                << "\n";
      std::cout << "  ---> child_offsets = " << (column->child_offsets[row_index + 1] + child_count)
                << "\n";
#endif
      column->string_lengths[row_index]    = column->child_offsets[row_index + 1] + child_count;
      column->child_offsets[row_index + 1] = column->child_offsets[row_index + 1] + child_count;
    };

  /**
   * @brief Gets the currently selected child column given a \p current_data_path.
   *
   * That is, if \p current_data_path top-of-stack is
   * (a) a struct, the selected child column corresponds to the child column of the last field name
   * node encountered.
   * (b) a list, the selected child column corresponds to single child column of
   * the list column. In this case, the child column may not exist yet.
   */
  auto get_selected_column = [](std::stack<tree_node>& current_data_path) {
    json_column* selected_col = current_data_path.top().current_selected_col;

    // If the node does not have a selected column yet
    if (selected_col == nullptr) {
      // We're looking at the child column of a list column
      if (current_data_path.top().column->type == json_col_t::ListColumn) {
        CUDF_EXPECTS(current_data_path.top().column->child_columns.size() <= 1,
                     "Encountered a list column with more than a single child column");
        // The child column has yet to be created
        if (current_data_path.top().column->child_columns.size() == 0) {
          current_data_path.top().column->child_columns.emplace(std::string{list_child_name},
                                                                json_column{json_col_t::Unknown});
          current_data_path.top().column->column_order.push_back(list_child_name);
        }
        current_data_path.top().current_selected_col =
          &current_data_path.top().column->child_columns.begin()->second;
        selected_col = current_data_path.top().current_selected_col;
      } else {
        CUDF_FAIL("Trying to retrieve child column without encountering a field name.");
      }
    }
#ifdef NJP_DEBUG_PRINT
    std::cout << "  -> get_selected_column()\n";
    std::cout << "  ---> selected col@" << selected_col << "\n";
#endif
    return selected_col;
  };

  /**
   * @brief Returns a pointer to the child column with the given \p field_name within the current
   * struct column.
   */
  auto select_column = [](std::stack<tree_node>& current_data_path, std::string const& field_name) {
#ifdef NJP_DEBUG_PRINT
    std::cout << "  -> select_column(" << field_name << ")\n";
#endif
    // The field name's parent struct node
    auto& current_struct_node = current_data_path.top();

    // Verify that the field name node is actually a child of a struct
    CUDF_EXPECTS(current_data_path.top().column->type == json_col_t::StructColumn,
                 "Invalid JSON token sequence");

    json_column* struct_col  = current_struct_node.column;
    auto const& child_col_it = struct_col->child_columns.find(field_name);

    // The field name's column exists already, select that as the struct node's currently selected
    // child column
    if (child_col_it != struct_col->child_columns.end()) { return &child_col_it->second; }

    // The field name's column does not exist yet, so we have to append the child column to the
    // struct column
    struct_col->column_order.push_back(field_name);
    return &struct_col->child_columns.emplace(field_name, json_column{}).first->second;
  };

  /**
   * @brief Gets the row offset at which to insert. I.e., for a child column of a list column, we
   * just have to append the row to the end. Otherwise we have to propagate the row offset from the
   * parent struct column.
   */
  auto get_target_row_index = [](std::stack<tree_node> const& current_data_path,
                                 json_column* target_column) {
#ifdef NJP_DEBUG_PRINT
    std::cout << " -> target row: "
              << ((current_data_path.top().column->type == json_col_t::ListColumn)
                    ? target_column->current_offset
                    : current_data_path.top().row_index)
              << "\n";
#endif
    return (current_data_path.top().column->type == json_col_t::ListColumn)
             ? target_column->current_offset
             : current_data_path.top().row_index;
  };

  // The offset of the token currently being processed
  std::size_t offset = 0;

  // Giving names to magic constants
  constexpr uint32_t zero_child_count = 0;

  CUDF_EXPECTS(tokens.size() == token_indices_gpu.size(),
               "Unexpected mismatch in number of token types and token indices");
  CUDF_EXPECTS(tokens.size() > 0, "Empty JSON input not supported");

  // The JSON root may only be a struct, list, string, or value node
  CUDF_EXPECTS(is_valid_root_token(tokens[offset]), "Invalid beginning of JSON document");

  while (offset < tokens.size()) {
    // Verify there's at least the JSON root node left on the stack to which we can append data
    CUDF_EXPECTS(current_data_path.size() > 0, "Invalid JSON structure");

    // Verify that the current node in the tree (which becomes this nodes parent) can have children
    CUDF_EXPECTS(current_data_path.top().column->type == json_col_t::ListColumn or
                   current_data_path.top().column->type == json_col_t::StructColumn,
                 "Invalid JSON structure");

    // The token we're currently parsing
    auto const& token = tokens[offset];

#ifdef NJP_DEBUG_PRINT
    std::cout << "[" << token_to_string(token) << "]\n";
#endif

    // StructBegin token
    if (token == token_t::StructBegin) {
      // Get this node's column. That is, the parent node's selected column:
      // (a) if parent is a list, then this will (create and) return the list's only child column
      // (b) if parent is a struct, then this will return the column selected by the last field name
      // encountered.
      json_column* selected_col = get_selected_column(current_data_path);

      // Get the row offset at which to insert
      auto const target_row_index = get_target_row_index(current_data_path, selected_col);

      // Increment parent's child count and insert this struct node into the data path
      current_data_path.top().num_children++;
      current_data_path.push({selected_col, target_row_index, nullptr, zero_child_count});

      // Add this struct node to the current column
      selected_col->append_row(target_row_index,
                               token_to_column_type(tokens[offset]),
                               get_token_index(tokens[offset], token_indices_gpu[offset]),
                               get_token_index(tokens[offset], token_indices_gpu[offset]),
                               zero_child_count);
    }

    // StructEnd token
    else if (token == token_t::StructEnd) {
      // Verify that this node in fact a struct node (i.e., it was part of a struct column)
      CUDF_EXPECTS(current_data_path.top().column->type == json_col_t::StructColumn,
                   "Broken invariant while parsing JSON");
      CUDF_EXPECTS(current_data_path.top().column != nullptr,
                   "Broken invariant while parsing JSON");

      // Update row to account for string offset
      update_row(current_data_path.top().column,
                 current_data_path.top().row_index,
                 get_token_index(tokens[offset], token_indices_gpu[offset]),
                 current_data_path.top().num_children);

      // Pop struct from the path stack
      current_data_path.pop();
    }

    // ListBegin token
    else if (token == token_t::ListBegin) {
      // Get the selected column
      json_column* selected_col = get_selected_column(current_data_path);

      // Get the row offset at which to insert
      auto const target_row_index = get_target_row_index(current_data_path, selected_col);

      // Increment parent's child count and insert this struct node into the data path
      current_data_path.top().num_children++;
      current_data_path.push({selected_col, target_row_index, nullptr, zero_child_count});

      // Add this struct node to the current column
      selected_col->append_row(target_row_index,
                               token_to_column_type(tokens[offset]),
                               get_token_index(tokens[offset], token_indices_gpu[offset]),
                               get_token_index(tokens[offset], token_indices_gpu[offset]),
                               zero_child_count);
    }

    // ListEnd token
    else if (token == token_t::ListEnd) {
      // Verify that this node in fact a list node (i.e., it was part of a list column)
      CUDF_EXPECTS(current_data_path.top().column->type == json_col_t::ListColumn,
                   "Broken invariant while parsing JSON");
      CUDF_EXPECTS(current_data_path.top().column != nullptr,
                   "Broken invariant while parsing JSON");

      // Update row to account for string offset
      update_row(current_data_path.top().column,
                 current_data_path.top().row_index,
                 get_token_index(tokens[offset], token_indices_gpu[offset]),
                 current_data_path.top().num_children);

      // Pop list from the path stack
      current_data_path.pop();
    }

    // Error token
    else if (token == token_t::ErrorBegin) {
#ifdef NJP_DEBUG_PRINT
      std::cout << "[ErrorBegin]\n";
      std::cout << "@" << get_token_index(tokens[offset], token_indices_gpu[offset]);
#endif
      CUDF_FAIL("Parser encountered an invalid format.");
    }

    // FieldName, String, or Value (begin, end)-pair
    else if (token == token_t::FieldNameBegin or token == token_t::StringBegin or
             token == token_t::ValueBegin) {
      // Verify that this token has the right successor to build a correct (being, end) token pair
      CUDF_EXPECTS((offset + 1) < tokens.size(), "Invalid JSON token sequence");
      CUDF_EXPECTS(tokens[offset + 1] == end_of_partner(token), "Invalid JSON token sequence");

      // The offset to the first symbol from the JSON input associated with the current token
      auto const& token_begin_offset = get_token_index(tokens[offset], token_indices_gpu[offset]);

      // The offset to one past the last symbol associated with the current token
      auto const& token_end_offset =
        get_token_index(tokens[offset + 1], token_indices_gpu[offset + 1]);

      // FieldNameBegin
      // For the current struct node in the tree, select the child column corresponding to this
      // field name
      if (token == token_t::FieldNameBegin) {
        std::string field_name{input.data() + token_begin_offset,
                               (token_end_offset - token_begin_offset)};
        current_data_path.top().current_selected_col = select_column(current_data_path, field_name);
      }
      // StringBegin
      // ValueBegin
      // As we currently parse to string columns there's no further differentiation
      else if (token == token_t::StringBegin or token == token_t::ValueBegin) {
        // Get the selected column
        json_column* selected_col = get_selected_column(current_data_path);

        // Get the row offset at which to insert
        auto const target_row_index = get_target_row_index(current_data_path, selected_col);

        current_data_path.top().num_children++;

        selected_col->append_row(target_row_index,
                                 token_to_column_type(token),
                                 token_begin_offset,
                                 token_end_offset,
                                 zero_child_count);
      } else {
        CUDF_FAIL("Unknown JSON token");
      }

      // As we've also consumed the end-of-* token, we advance the processed token offset by one
      offset++;
    }

    offset++;
  }

  // Make sure all of a struct's child columns have the same length
  root_column.level_child_cols_recursively(root_column.current_offset);
}

/**
 * @brief Retrieves the parse_options to be used for type inference and type casting
 *
 * @param options The reader options to influence the relevant type inference and type casting
 * options
 */
auto parsing_options(cudf::io::json_reader_options const& options)
{
  auto parse_opts = cudf::io::parse_options{',', '\n', '\"', '.'};

  auto const stream     = cudf::get_default_stream();
  parse_opts.dayfirst   = options.is_enabled_dayfirst();
  parse_opts.keepquotes = options.is_enabled_keep_quotes();
  parse_opts.trie_true  = cudf::detail::create_serialized_trie({"true"}, stream);
  parse_opts.trie_false = cudf::detail::create_serialized_trie({"false"}, stream);
  parse_opts.trie_na    = cudf::detail::create_serialized_trie({"", "null"}, stream);
  return parse_opts;
}

std::pair<std::unique_ptr<column>, std::vector<column_name_info>> json_column_to_cudf_column(
  json_column const& json_col,
  device_span<SymbolT const> d_input,
  cudf::io::json_reader_options const& options,
  std::optional<schema_element> schema,
  rmm::cuda_stream_view stream,
  rmm::mr::device_memory_resource* mr)
{
  // Range of orchestrating/encapsulating function
  CUDF_FUNC_RANGE();

  auto make_validity =
    [stream, mr](json_column const& json_col) -> std::pair<rmm::device_buffer, size_type> {
    return {rmm::device_buffer{json_col.validity.data(),
                               bitmask_allocation_size_bytes(json_col.current_offset),
                               stream,
                               mr},
            json_col.current_offset - json_col.valid_count};
  };

  auto get_child_schema = [schema](auto child_name) -> std::optional<schema_element> {
    if (schema.has_value()) {
      auto const result = schema.value().child_types.find(child_name);
      if (result != std::end(schema.value().child_types)) { return result->second; }
    }
    return {};
  };

  switch (json_col.type) {
    case json_col_t::StringColumn: {
      auto const col_size = json_col.string_offsets.size();
      CUDF_EXPECTS(json_col.string_offsets.size() == json_col.string_lengths.size(),
                   "string offset, string length mismatch");

      // Move string_offsets and string_lengths to GPU
      rmm::device_uvector<json_column::row_offset_t> d_string_offsets =
        cudf::detail::make_device_uvector_async(json_col.string_offsets, stream);
      rmm::device_uvector<json_column::row_offset_t> d_string_lengths =
        cudf::detail::make_device_uvector_async(json_col.string_lengths, stream);

      // Prepare iterator that returns (string_offset, string_length)-tuples
      auto offset_length_it =
        thrust::make_zip_iterator(d_string_offsets.begin(), d_string_lengths.begin());

      // Prepare iterator that returns (string_offset, string_length)-pairs needed by inference
      auto string_ranges_it =
        thrust::make_transform_iterator(offset_length_it, [] __device__(auto ip) {
          return thrust::pair<json_column::row_offset_t, std::size_t>{
            thrust::get<0>(ip), static_cast<std::size_t>(thrust::get<1>(ip))};
        });

      // Prepare iterator that returns (string_ptr, string_length)-pairs needed by type conversion
      auto string_spans_it = thrust::make_transform_iterator(
        offset_length_it, [data = d_input.data()] __device__(auto ip) {
          return thrust::pair<const char*, std::size_t>{
            data + thrust::get<0>(ip), static_cast<std::size_t>(thrust::get<1>(ip))};
        });

      data_type target_type{};

      if (schema.has_value()) {
#ifdef NJP_DEBUG_PRINT
        std::cout << "-> explicit type: "
                  << (schema.has_value() ? std::to_string(static_cast<int>(schema->type.id()))
                                         : "n/a");
#endif
        target_type = schema.value().type;
      }
      // Infer column type, if we don't have an explicit type for it
      else {
        target_type = cudf::io::detail::infer_data_type(
          parsing_options(options).json_view(), d_input, string_ranges_it, col_size, stream);
      }

      // Convert strings to the inferred data type
      auto col = experimental::detail::parse_data(string_spans_it,
                                                  col_size,
                                                  target_type,
                                                  make_validity(json_col).first,
                                                  parsing_options(options).view(),
                                                  stream,
                                                  mr);

      // Reset nullable if we do not have nulls
      // This is to match the existing JSON reader's behaviour:
      // - Non-string columns will always be returned as nullable
      // - String columns will be returned as nullable, iff there's at least one null entry
      if (target_type.id() == type_id::STRING and col->null_count() == 0) {
        col->set_null_mask(rmm::device_buffer{0, stream, mr}, 0);
      }

      // For string columns return ["offsets", "char"] schema
      if (target_type.id() == type_id::STRING) {
        return {std::move(col), {{"offsets"}, {"chars"}}};
      }
      // Non-string leaf-columns (e.g., numeric) do not have child columns in the schema
      else {
        return {std::move(col), {}};
      }
      break;
    }
    case json_col_t::StructColumn: {
      std::vector<std::unique_ptr<column>> child_columns;
      std::vector<column_name_info> column_names{};
      size_type num_rows{json_col.current_offset};
      // Create children columns
      for (auto const& col_name : json_col.column_order) {
        auto const& col = json_col.child_columns.find(col_name);
        column_names.emplace_back(col->first);
        auto const& child_col      = col->second;
        auto [child_column, names] = json_column_to_cudf_column(
          child_col, d_input, options, get_child_schema(col_name), stream, mr);
        CUDF_EXPECTS(num_rows == child_column->size(),
                     "All children columns must have the same size");
        child_columns.push_back(std::move(child_column));
        column_names.back().children = names;
      }
      auto [result_bitmask, null_count] = make_validity(json_col);
      return {
        make_structs_column(
          num_rows, std::move(child_columns), null_count, std::move(result_bitmask), stream, mr),
        column_names};
      break;
    }
    case json_col_t::ListColumn: {
      size_type num_rows = json_col.child_offsets.size();
      std::vector<column_name_info> column_names{};
      column_names.emplace_back("offsets");
      column_names.emplace_back(
        json_col.child_columns.empty() ? list_child_name : json_col.child_columns.begin()->first);

      rmm::device_uvector<json_column::row_offset_t> d_offsets =
        cudf::detail::make_device_uvector_async(json_col.child_offsets, stream, mr);
      auto offsets_column =
        std::make_unique<column>(data_type{type_id::INT32}, num_rows, d_offsets.release());
      // Create children column
      auto [child_column, names] =
        json_col.child_columns.empty()
          ? std::pair<std::unique_ptr<column>,
                      std::vector<column_name_info>>{std::make_unique<column>(), {}}
          : json_column_to_cudf_column(json_col.child_columns.begin()->second,
                                       d_input,
                                       options,
                                       get_child_schema(json_col.child_columns.begin()->first),
                                       stream,
                                       mr);
      column_names.back().children      = names;
      auto [result_bitmask, null_count] = make_validity(json_col);
      return {make_lists_column(num_rows - 1,
                                std::move(offsets_column),
                                std::move(child_column),
                                null_count,
                                std::move(result_bitmask),
                                stream,
                                mr),
              std::move(column_names)};
      break;
    }
    default: CUDF_FAIL("Unsupported column type, yet to be implemented"); break;
  }

  return {};
}

table_with_metadata host_parse_nested_json(host_span<SymbolT const> input,
                                           cudf::io::json_reader_options const& options,
                                           rmm::cuda_stream_view stream,
                                           rmm::mr::device_memory_resource* mr)
{
  // Range of orchestrating/encapsulating function
  CUDF_FUNC_RANGE();

  auto const new_line_delimited_json = options.is_enabled_lines();

  // Allocate device memory for the JSON input & copy over to device
  rmm::device_uvector<SymbolT> d_input = cudf::detail::make_device_uvector_async(input, stream);

  // Get internal JSON column
  json_column root_column{};
  std::stack<tree_node> data_path{};

  constexpr uint32_t row_offset_zero            = 0;
  constexpr uint32_t token_begin_offset_zero    = 0;
  constexpr uint32_t token_end_offset_zero      = 0;
  constexpr uint32_t node_init_child_count_zero = 0;

  // Whether the tokenizer stage should keep quote characters for string values
  // If the tokenizer keeps the quote characters, they may be stripped during type casting
  constexpr bool include_quote_chars = true;

  // We initialize the very root node and root column, which represent the JSON document being
  // parsed. That root node is a list node and that root column is a list column. The column has the
  // root node as its only row. The values parsed from the JSON input will be treated as follows:
  // (1) For JSON lines: we expect to find a list of JSON values that all
  // will be inserted into this root list column. (2) For regular JSON: we expect to have only a
  // single value (list, struct, string, number, literal) that will be inserted into this root
  // column.
  root_column.append_row(
    row_offset_zero, json_col_t::ListColumn, token_begin_offset_zero, token_end_offset_zero, 1);

  // Push the root node onto the stack for the data path
  data_path.push({&root_column, row_offset_zero, nullptr, node_init_child_count_zero});

  make_json_column(
    root_column, data_path, input, d_input, options, include_quote_chars, stream, mr);

  // data_root refers to the root column of the data represented by the given JSON string
  auto const& data_root =
    new_line_delimited_json ? root_column : root_column.child_columns.begin()->second;

  // Zero row entries
  if (data_root.type == json_col_t::ListColumn && data_root.child_columns.size() == 0) {
    return table_with_metadata{std::make_unique<table>(std::vector<std::unique_ptr<column>>{}),
                               {{}, std::vector<column_name_info>{}}};
  }

  // Verify that we were in fact given a list of structs (or in JSON speech: an array of objects)
  auto constexpr single_child_col_count = 1;
  CUDF_EXPECTS(data_root.type == json_col_t::ListColumn and
                 data_root.child_columns.size() == single_child_col_count and
                 data_root.child_columns.begin()->second.type == json_col_t::StructColumn,
               "Currently the nested JSON parser only supports an array of (nested) objects");

  // Slice off the root list column, which has only a single row that contains all the structs
  auto const& root_struct_col = data_root.child_columns.begin()->second;

  // Initialize meta data to be populated while recursing through the tree of columns
  std::vector<std::unique_ptr<column>> out_columns;
  std::vector<column_name_info> out_column_names;

  // Iterate over the struct's child columns and convert to cudf column
  size_type column_index = 0;
  for (auto const& col_name : root_struct_col.column_order) {
    auto const& json_col = root_struct_col.child_columns.find(col_name)->second;
    // Insert this columns name into the schema
    out_column_names.emplace_back(col_name);

    std::optional<schema_element> child_schema_element = std::visit(
      cudf::detail::visitor_overload{
        [column_index](const std::vector<data_type>& user_dtypes) -> std::optional<schema_element> {
          auto ret = (static_cast<std::size_t>(column_index) < user_dtypes.size())
                       ? std::optional<schema_element>{{user_dtypes[column_index]}}
                       : std::optional<schema_element>{};
#ifdef NJP_DEBUG_PRINT
          std::cout << "Column by index: #" << column_index << ", type id: "
                    << (ret.has_value() ? std::to_string(static_cast<int>(ret->type.id())) : "n/a")
                    << ", with " << (ret.has_value() ? ret->child_types.size() : 0) << " children"
                    << "\n";
#endif
          return ret;
        },
        [col_name](
          std::map<std::string, data_type> const& user_dtypes) -> std::optional<schema_element> {
          auto ret = (user_dtypes.find(col_name) != std::end(user_dtypes))
                       ? std::optional<schema_element>{{user_dtypes.find(col_name)->second}}
                       : std::optional<schema_element>{};
#ifdef NJP_DEBUG_PRINT
          std::cout << "Column by flat name: '" << col_name << "', type id: "
                    << (ret.has_value() ? std::to_string(static_cast<int>(ret->type.id())) : "n/a")
                    << ", with " << (ret.has_value() ? ret->child_types.size() : 0) << " children"
                    << "\n";
#endif
          return ret;
        },
        [col_name](std::map<std::string, schema_element> const& user_dtypes)
          -> std::optional<schema_element> {
          auto ret = (user_dtypes.find(col_name) != std::end(user_dtypes))
                       ? user_dtypes.find(col_name)->second
                       : std::optional<schema_element>{};
#ifdef NJP_DEBUG_PRINT
          std::cout << "Column by nested name: #" << col_name << ", type id: "
                    << (ret.has_value() ? std::to_string(static_cast<int>(ret->type.id())) : "n/a")
                    << ", with " << (ret.has_value() ? ret->child_types.size() : 0) << " children"
                    << "\n";
#endif
          return ret;
        }},
      options.get_dtypes());

    // Get this JSON column's cudf column and schema info
    auto [cudf_col, col_name_info] =
      json_column_to_cudf_column(json_col, d_input, options, child_schema_element, stream, mr);
    out_column_names.back().children = std::move(col_name_info);
    out_columns.emplace_back(std::move(cudf_col));

    column_index++;
  }

  return table_with_metadata{std::make_unique<table>(std::move(out_columns)),
                             {{}, out_column_names}};
}

}  // namespace detail
}  // namespace cudf::io::json

// Debug print flag
#undef NJP_DEBUG_PRINT
