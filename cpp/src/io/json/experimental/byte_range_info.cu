/*
 * Copyright (c) 2022, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <cudf/types.hpp>
#include <cudf/utilities/span.hpp>

#include <rmm/cuda_stream_view.hpp>
#include <rmm/exec_policy.hpp>
#include <thrust/find.h>

namespace cudf::io::detail::json::experimental {

// Extract the first character position in the string.
size_type find_first_delimiter(device_span<char const> d_data,
                               char const delimiter,
                               rmm::cuda_stream_view stream)
{
  auto const first_delimiter_position =
    thrust::find(rmm::exec_policy(stream), d_data.begin(), d_data.end(), delimiter);
  return first_delimiter_position != d_data.end() ? first_delimiter_position - d_data.begin() : -1;
}

}  // namespace cudf::io::detail::json::experimental
