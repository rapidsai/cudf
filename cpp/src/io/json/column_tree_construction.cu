#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2024, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "json_utils.hpp"
#include "nested_json.hpp"

#include <cudf/column/column_factories.hpp>
#include <cudf/detail/null_mask.hpp>
#include <cudf/detail/nvtx/ranges.hpp>
#include <cudf/detail/utilities/vector_factories.hpp>
#include <cudf/detail/utilities/visitor_overload.hpp>
#include <cudf/strings/strings_column_view.hpp>
#include <cudf/types.hpp>
#include <cudf/utilities/error.hpp>
#include <cudf/utilities/span.hpp>

#include <rmm/cuda_stream_view.hpp>
#include <rmm/device_uvector.hpp>
#include <rmm/exec_policy.hpp>
#include <rmm/resource_ref.hpp>

#include <cub/device/device_segmented_reduce.cuh>
#include <cuda/atomic>
#include <cuda/functional>
#include <thrust/count.h>
#include <thrust/for_each.h>
#include <thrust/functional.h>
#include <thrust/gather.h>
#include <thrust/iterator/constant_iterator.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/iterator/discard_iterator.h>
#include <thrust/iterator/permutation_iterator.h>
#include <thrust/iterator/transform_output_iterator.h>
#include <thrust/iterator/zip_iterator.h>
#include <thrust/reduce.h>
#include <thrust/scan.h>
#include <thrust/sort.h>
#include <thrust/transform.h>
#include <thrust/transform_scan.h>
#include <thrust/unique.h>

namespace cudf::io::json {

template <typename T>
void print(device_span<T const> d_vec, std::string name, rmm::cuda_stream_view stream)
{
  stream.synchronize();
  auto h_vec = cudf::detail::make_std_vector_async(d_vec, stream);
  stream.synchronize();
  std::cout << name << " = ";
  for (auto e : h_vec) {
    std::cout << e << " ";
  }
  std::cout << std::endl;
}

namespace experimental::detail {

using row_offset_t = size_type;

struct level_ordering {
  device_span<TreeDepthT> node_levels;
  device_span<NodeIndexT> col_ids;
  __device__ bool operator()(NodeIndexT lhs_node_id, NodeIndexT rhs_node_id) const
  {
    return (node_levels[lhs_node_id] < node_levels[rhs_node_id]) ||
           (node_levels[lhs_node_id] == node_levels[rhs_node_id] &&
            col_ids[lhs_node_id] < col_ids[rhs_node_id]);
  }
};

struct parent_nodeids_to_colids {
  device_span<NodeIndexT> col_ids;
  device_span<NodeIndexT> rev_mapped_col_ids;
  __device__ auto operator()(NodeIndexT parent_node_id) -> NodeIndexT
  {
    return parent_node_id == parent_node_sentinel ? parent_node_sentinel
                                                  : rev_mapped_col_ids[col_ids[parent_node_id]];
  }
};

/**
 * @brief Reduces node tree representation to column tree CSR representation.
 *
 * @param tree Node tree representation of JSON string
 * @param original_col_ids Column ids of nodes
 * @param sorted_col_ids Sorted column ids of nodes
 * @param ordered_node_ids Node ids of nodes sorted by column ids
 * @param row_offsets Row offsets of nodes
 * @param is_array_of_arrays Whether the tree is an array of arrays
 * @param row_array_parent_col_id Column id of row array, if is_array_of_arrays is true
 * @param stream CUDA stream used for device memory operations and kernel launches
 * @return A tuple of column tree representation of JSON string, column ids of columns, and
 * max row offsets of columns
 */
std::tuple<csr, column_tree_properties> reduce_to_column_tree(
  tree_meta_t& tree,
  device_span<NodeIndexT> col_ids,
  device_span<row_offset_t> row_offsets,
  bool is_array_of_arrays,
  NodeIndexT const row_array_parent_col_id,
  rmm::cuda_stream_view stream)
{
  CUDF_FUNC_RANGE();

  rmm::device_uvector<NodeIndexT> level_ordered_col_ids(col_ids.size(), stream);
  rmm::device_uvector<NodeIndexT> level_ordered_node_ids(col_ids.size(), stream);
  thrust::copy(
    rmm::exec_policy_nosync(stream), col_ids.begin(), col_ids.end(), level_ordered_col_ids.begin());
  thrust::sequence(
    rmm::exec_policy_nosync(stream), level_ordered_node_ids.begin(), level_ordered_node_ids.end());

  // Reorder nodes and column ids in level-wise fashion
  thrust::stable_sort_by_key(rmm::exec_policy_nosync(stream),
                             level_ordered_node_ids.begin(),
                             level_ordered_node_ids.end(),
                             level_ordered_col_ids.begin(),
                             level_ordering{tree.node_levels, col_ids});

  // 1. get the number of columns in tree, mapping between node tree col ids and csr col ids, and
  // the node id of first row in each column
  auto const num_columns = thrust::unique_count(
    rmm::exec_policy_nosync(stream), level_ordered_col_ids.begin(), level_ordered_col_ids.end());
  rmm::device_uvector<NodeIndexT> level_ordered_unique_node_ids(num_columns, stream);
  rmm::device_uvector<NodeIndexT> mapped_col_ids(num_columns, stream);
  rmm::device_uvector<NodeIndexT> rev_mapped_col_ids(num_columns, stream);
  thrust::unique_by_key_copy(rmm::exec_policy_nosync(stream),
                             level_ordered_col_ids.begin(),
                             level_ordered_node_ids.end(),
                             level_ordered_node_ids.begin(),
                             mapped_col_ids.begin(),
                             level_ordered_unique_node_ids.begin());
  auto* dev_num_levels_ptr = thrust::max_element(
    rmm::exec_policy_nosync(stream), tree.node_levels.begin(), tree.node_levels.end());

  rmm::device_uvector<NodeIndexT> mapped_col_ids_copy(num_columns, stream);
  thrust::copy(rmm::exec_policy_nosync(stream),
               mapped_col_ids.begin(),
               mapped_col_ids.end(),
               mapped_col_ids_copy.begin());
  thrust::sequence(
    rmm::exec_policy_nosync(stream), rev_mapped_col_ids.begin(), rev_mapped_col_ids.end());
  thrust::sort_by_key(rmm::exec_policy_nosync(stream),
                      mapped_col_ids_copy.begin(),
                      mapped_col_ids_copy.end(),
                      rev_mapped_col_ids.begin());

  // 2. maximum number of rows per column: computed with reduce_by_key {col_id}, {row_offset}, max.
  // 3. category for each column node by aggregating all nodes in node tree corresponding to same
  // column:
  //    reduce_by_key {col_id}, {node_categories} - custom opp (*+v=*, v+v=v, *+#=E)
  rmm::device_uvector<row_offset_t> max_row_offsets(num_columns, stream);
  rmm::device_uvector<NodeT> column_categories(num_columns, stream);
  auto ordered_row_offsets =
    thrust::make_permutation_iterator(row_offsets.begin(), level_ordered_node_ids.begin());
  auto ordered_node_categories =
    thrust::make_permutation_iterator(tree.node_categories.begin(), level_ordered_node_ids.begin());
  thrust::reduce_by_key(
    rmm::exec_policy_nosync(stream),
    level_ordered_col_ids.begin(),
    level_ordered_col_ids.end(),
    thrust::make_zip_iterator(ordered_row_offsets, ordered_node_categories),
    thrust::make_discard_iterator(),
    thrust::make_zip_iterator(max_row_offsets.begin(), column_categories.begin()),
    thrust::equal_to<NodeIndexT>(),
    [] __device__(auto a, auto b) {
      auto row_offset_a = thrust::get<0>(a);
      auto row_offset_b = thrust::get<0>(b);
      auto type_a       = thrust::get<1>(a);
      auto type_b       = thrust::get<1>(b);

      NodeT ctg;
      auto is_a_leaf = (type_a == NC_VAL || type_a == NC_STR);
      auto is_b_leaf = (type_b == NC_VAL || type_b == NC_STR);
      // (v+v=v, *+*=*,  *+v=*, *+#=E, NESTED+VAL=NESTED)
      // *+*=*, v+v=v
      if (type_a == type_b) {
        ctg = type_a;
      } else if (is_a_leaf) {
        // *+v=*, N+V=N
        // STRUCT/LIST + STR/VAL = STRUCT/LIST, STR/VAL + FN = ERR, STR/VAL + STR = STR
        ctg = (type_b == NC_FN ? NC_ERR : (is_b_leaf ? NC_STR : type_b));
      } else if (is_b_leaf) {
        ctg = (type_a == NC_FN ? NC_ERR : (is_a_leaf ? NC_STR : type_a));
      } else
        ctg = NC_ERR;

      thrust::maximum<row_offset_t> row_offset_op;
      return thrust::make_pair(row_offset_op(row_offset_a, row_offset_b), ctg);
    });

  // 4. construct parent_col_ids using permutation iterator
  rmm::device_uvector<NodeIndexT> parent_col_ids(num_columns, stream);
  thrust::transform_output_iterator parent_col_ids_it(
    parent_col_ids.begin(), parent_nodeids_to_colids{col_ids, rev_mapped_col_ids});
  thrust::copy_n(rmm::exec_policy_nosync(stream),
                 thrust::make_permutation_iterator(tree.parent_node_ids.begin(),
                                                   level_ordered_unique_node_ids.begin()),
                 num_columns,
                 parent_col_ids_it);

  /*
    5. CSR construction:
      a. Sort column levels and get their ordering
      b. For each column node coln iterated according to sorted_column_levels; do
          i. Find nodes that have coln as the parent node -> set adj_coln
          ii. row idx[coln] = size of adj_coln + 1
          iii. col idx[coln] = adj_coln U {parent_col_id[coln]}
  */

  rmm::device_uvector<NodeIndexT> rowidx(num_columns + 1, stream);
  thrust::fill(rmm::exec_policy_nosync(stream), rowidx.begin(), rowidx.end(), 0);
  // Note that the first element of csr_parent_col_ids is -1 (parent_node_sentinel)
  // children adjacency
  auto num_non_leaf_columns = thrust::unique_count(
    rmm::exec_policy_nosync(stream), parent_col_ids.begin() + 1, parent_col_ids.end());
  thrust::reduce_by_key(rmm::exec_policy_nosync(stream),
                        parent_col_ids.begin() + 1,
                        parent_col_ids.end(),
                        thrust::make_constant_iterator(1),
                        thrust::make_discard_iterator(),
                        rowidx.begin() + 1,
                        thrust::equal_to<TreeDepthT>());
  thrust::transform_inclusive_scan(
    rmm::exec_policy_nosync(stream),
    thrust::make_zip_iterator(thrust::make_counting_iterator(1), rowidx.begin() + 1),
    thrust::make_zip_iterator(thrust::make_counting_iterator(1) + num_columns, rowidx.end()),
    rowidx.begin() + 1,
    cuda::proclaim_return_type<NodeIndexT>([] __device__(auto a) {
      auto n   = thrust::get<0>(a);
      auto idx = thrust::get<1>(a);
      return n == 1 ? idx : idx + 1;
    }),
    thrust::plus<NodeIndexT>{});

  rmm::device_uvector<NodeIndexT> colidx((num_columns - 1) * 2, stream);
  // Skip the parent of root node
  thrust::scatter(rmm::exec_policy_nosync(stream),
                  parent_col_ids.begin() + 1,
                  parent_col_ids.end(),
                  rowidx.begin() + 1,
                  colidx.begin());
  // excluding root node, construct scatter map
  rmm::device_uvector<NodeIndexT> map(num_columns - 1, stream);
  thrust::inclusive_scan_by_key(rmm::exec_policy_nosync(stream),
                                parent_col_ids.begin() + 1,
                                parent_col_ids.end(),
                                thrust::make_constant_iterator(1),
                                map.begin());
  thrust::for_each_n(rmm::exec_policy_nosync(stream),
                     thrust::make_counting_iterator(1),
                     num_columns - 1,
                     [rowidx         = rowidx.begin(),
                      map            = map.begin(),
                      parent_col_ids = parent_col_ids.begin()] __device__(auto i) {
                       auto parent_col_id = parent_col_ids[i];
                       if (parent_col_id == 0)
                         map[i - 1]--;
                       else
                         map[i - 1] += rowidx[parent_col_id];
                     });
  thrust::scatter(rmm::exec_policy_nosync(stream),
                  thrust::make_counting_iterator(1),
                  thrust::make_counting_iterator(1) + num_columns - 1,
                  map.begin(),
                  colidx.begin());

  // Mixed types in List children go to different columns,
  // so all immediate children of list column should have same max_row_offsets.
  //   create list's children max_row_offsets array
  //   gather the max_row_offsets from children row offset array.
  {
    auto max_row_offsets_it =
      thrust::make_permutation_iterator(max_row_offsets.begin(), colidx.begin());
    rmm::device_uvector<row_offset_t> max_children_max_row_offsets(num_columns, stream);
    size_t temp_storage_bytes = 0;
    hipcub::DeviceSegmentedReduce::Max(nullptr,
                                    temp_storage_bytes,
                                    max_row_offsets_it,
                                    max_children_max_row_offsets.begin(),
                                    num_columns,
                                    rowidx.begin(),
                                    rowidx.begin() + 1,
                                    stream.value());
    rmm::device_buffer d_temp_storage(temp_storage_bytes, stream);
    hipcub::DeviceSegmentedReduce::Max(d_temp_storage.data(),
                                    temp_storage_bytes,
                                    max_row_offsets_it,
                                    max_children_max_row_offsets.begin(),
                                    num_columns,
                                    rowidx.begin(),
                                    rowidx.begin() + 1,
                                    stream.value());

    rmm::device_uvector<NodeIndexT> list_ancestors(num_columns, stream);
    thrust::for_each_n(
      rmm::exec_policy_nosync(stream),
      thrust::make_counting_iterator(0),
      num_columns,
      [rowidx            = rowidx.begin(),
       colidx            = colidx.begin(),
       column_categories = column_categories.begin(),
       dev_num_levels_ptr,
       list_ancestors = list_ancestors.begin()] __device__(NodeIndexT node) {
        auto num_levels      = *dev_num_levels_ptr;
        list_ancestors[node] = node;
        for (int level = 0; level <= num_levels; level++) {
          if (list_ancestors[node] > 0) list_ancestors[node] = colidx[rowidx[list_ancestors[node]]];
          if (list_ancestors[node] == 0 || column_categories[list_ancestors[node]] == NC_LIST)
            break;
        }
      });

    thrust::gather_if(rmm::exec_policy_nosync(stream),
                      list_ancestors.begin(),
                      list_ancestors.end(),
                      list_ancestors.begin(),
                      max_children_max_row_offsets.begin(),
                      max_row_offsets.begin(),
                      [] __device__(auto ancestor) { return ancestor != -1; });
  }

  return std::tuple{
    csr{std::move(rowidx), std::move(colidx)},
    column_tree_properties{
      std::move(column_categories), std::move(max_row_offsets), std::move(mapped_col_ids)}};
}

}  // namespace experimental::detail

namespace detail {
/**
 * @brief Reduces node tree representation to column tree representation.
 *
 * @param tree Node tree representation of JSON string
 * @param original_col_ids Column ids of nodes
 * @param sorted_col_ids Sorted column ids of nodes
 * @param ordered_node_ids Node ids of nodes sorted by column ids
 * @param row_offsets Row offsets of nodes
 * @param is_array_of_arrays Whether the tree is an array of arrays
 * @param row_array_parent_col_id Column id of row array, if is_array_of_arrays is true
 * @param stream CUDA stream used for device memory operations and kernel launches
 * @return A tuple of column tree representation of JSON string, column ids of columns, and
 * max row offsets of columns
 */
std::tuple<tree_meta_t, rmm::device_uvector<NodeIndexT>, rmm::device_uvector<size_type>>
reduce_to_column_tree(tree_meta_t& tree,
                      device_span<NodeIndexT> original_col_ids,
                      device_span<NodeIndexT> sorted_col_ids,
                      device_span<NodeIndexT> ordered_node_ids,
                      device_span<size_type> row_offsets,
                      bool is_array_of_arrays,
                      NodeIndexT const row_array_parent_col_id,
                      rmm::cuda_stream_view stream)
{
  CUDF_FUNC_RANGE();
  // 1. column count for allocation
  auto const num_columns =
    thrust::unique_count(rmm::exec_policy(stream), sorted_col_ids.begin(), sorted_col_ids.end());

  // 2. reduce_by_key {col_id}, {row_offset}, max.
  rmm::device_uvector<NodeIndexT> unique_col_ids(num_columns, stream);
  rmm::device_uvector<size_type> max_row_offsets(num_columns, stream);
  auto ordered_row_offsets =
    thrust::make_permutation_iterator(row_offsets.begin(), ordered_node_ids.begin());
  thrust::reduce_by_key(rmm::exec_policy(stream),
                        sorted_col_ids.begin(),
                        sorted_col_ids.end(),
                        ordered_row_offsets,
                        unique_col_ids.begin(),
                        max_row_offsets.begin(),
                        thrust::equal_to<size_type>(),
                        thrust::maximum<size_type>());

  // 3. reduce_by_key {col_id}, {node_categories} - custom opp (*+v=*, v+v=v, *+#=E)
  rmm::device_uvector<NodeT> column_categories(num_columns, stream);
  thrust::reduce_by_key(
    rmm::exec_policy(stream),
    sorted_col_ids.begin(),
    sorted_col_ids.end(),
    thrust::make_permutation_iterator(tree.node_categories.begin(), ordered_node_ids.begin()),
    unique_col_ids.begin(),
    column_categories.begin(),
    thrust::equal_to<size_type>(),
    [] __device__(NodeT type_a, NodeT type_b) -> NodeT {
      auto is_a_leaf = (type_a == NC_VAL || type_a == NC_STR);
      auto is_b_leaf = (type_b == NC_VAL || type_b == NC_STR);
      // (v+v=v, *+*=*,  *+v=*, *+#=E, NESTED+VAL=NESTED)
      // *+*=*, v+v=v
      if (type_a == type_b) {
        return type_a;
      } else if (is_a_leaf) {
        // *+v=*, N+V=N
        // STRUCT/LIST + STR/VAL = STRUCT/LIST, STR/VAL + FN = ERR, STR/VAL + STR = STR
        return type_b == NC_FN ? NC_ERR : (is_b_leaf ? NC_STR : type_b);
      } else if (is_b_leaf) {
        return type_a == NC_FN ? NC_ERR : (is_a_leaf ? NC_STR : type_a);
      }
      // *+#=E
      return NC_ERR;
    });

  // 4. unique_copy parent_node_ids, ranges
  rmm::device_uvector<TreeDepthT> column_levels(0, stream);  // not required
  rmm::device_uvector<NodeIndexT> parent_col_ids(num_columns, stream);
  rmm::device_uvector<SymbolOffsetT> col_range_begin(num_columns, stream);  // Field names
  rmm::device_uvector<SymbolOffsetT> col_range_end(num_columns, stream);
  rmm::device_uvector<size_type> unique_node_ids(num_columns, stream);
  thrust::unique_by_key_copy(rmm::exec_policy(stream),
                             sorted_col_ids.begin(),
                             sorted_col_ids.end(),
                             ordered_node_ids.begin(),
                             thrust::make_discard_iterator(),
                             unique_node_ids.begin());
  thrust::copy_n(
    rmm::exec_policy(stream),
    thrust::make_zip_iterator(
      thrust::make_permutation_iterator(tree.parent_node_ids.begin(), unique_node_ids.begin()),
      thrust::make_permutation_iterator(tree.node_range_begin.begin(), unique_node_ids.begin()),
      thrust::make_permutation_iterator(tree.node_range_end.begin(), unique_node_ids.begin())),
    unique_node_ids.size(),
    thrust::make_zip_iterator(
      parent_col_ids.begin(), col_range_begin.begin(), col_range_end.begin()));

  // convert parent_node_ids to parent_col_ids
  thrust::transform(
    rmm::exec_policy(stream),
    parent_col_ids.begin(),
    parent_col_ids.end(),
    parent_col_ids.begin(),
    [col_ids = original_col_ids.begin()] __device__(auto parent_node_id) -> size_type {
      return parent_node_id == parent_node_sentinel ? parent_node_sentinel
                                                    : col_ids[parent_node_id];
    });

  // condition is true if parent is not a list, or sentinel/root
  // Special case to return true if parent is a list and is_array_of_arrays is true
  auto is_non_list_parent = [column_categories = column_categories.begin(),
                             is_array_of_arrays,
                             row_array_parent_col_id] __device__(auto parent_col_id) -> bool {
    return !(parent_col_id == parent_node_sentinel ||
             column_categories[parent_col_id] == NC_LIST &&
               (!is_array_of_arrays || parent_col_id != row_array_parent_col_id));
    return (parent_col_id != parent_node_sentinel) &&
             (column_categories[parent_col_id] != NC_LIST) ||
           (is_array_of_arrays == true && parent_col_id == row_array_parent_col_id);
  };

  // Mixed types in List children go to different columns,
  // so all immediate children of list column should have same max_row_offsets.
  //   create list's children max_row_offsets array. (initialize to zero)
  //   atomicMax on  children max_row_offsets array.
  //   gather the max_row_offsets from children row offset array.
  {
    rmm::device_uvector<NodeIndexT> list_parents_children_max_row_offsets(num_columns, stream);
    thrust::fill(rmm::exec_policy(stream),
                 list_parents_children_max_row_offsets.begin(),
                 list_parents_children_max_row_offsets.end(),
                 0);
    thrust::for_each(rmm::exec_policy(stream),
                     unique_col_ids.begin(),
                     unique_col_ids.end(),
                     [column_categories = column_categories.begin(),
                      parent_col_ids    = parent_col_ids.begin(),
                      max_row_offsets   = max_row_offsets.begin(),
                      list_parents_children_max_row_offsets =
                        list_parents_children_max_row_offsets.begin()] __device__(auto col_id) {
                       auto parent_col_id = parent_col_ids[col_id];
                       if (parent_col_id != parent_node_sentinel and
                           column_categories[parent_col_id] == node_t::NC_LIST) {
                         cuda::atomic_ref<NodeIndexT, cuda::thread_scope_device> ref{
                           *(list_parents_children_max_row_offsets + parent_col_id)};
                         ref.fetch_max(max_row_offsets[col_id], cuda::std::memory_order_relaxed);
                       }
                     });
    thrust::gather_if(
      rmm::exec_policy(stream),
      parent_col_ids.begin(),
      parent_col_ids.end(),
      parent_col_ids.begin(),
      list_parents_children_max_row_offsets.begin(),
      max_row_offsets.begin(),
      [column_categories = column_categories.begin()] __device__(size_type parent_col_id) {
        return parent_col_id != parent_node_sentinel and
               column_categories[parent_col_id] == node_t::NC_LIST;
      });
  }

  // copy lists' max_row_offsets to children.
  // all structs should have same size.
  thrust::transform_if(
    rmm::exec_policy(stream),
    unique_col_ids.begin(),
    unique_col_ids.end(),
    max_row_offsets.begin(),
    [column_categories = column_categories.begin(),
     is_non_list_parent,
     parent_col_ids  = parent_col_ids.begin(),
     max_row_offsets = max_row_offsets.begin()] __device__(size_type col_id) {
      auto parent_col_id = parent_col_ids[col_id];
      // condition is true if parent is not a list, or sentinel/root
      while (is_non_list_parent(parent_col_id)) {
        col_id        = parent_col_id;
        parent_col_id = parent_col_ids[parent_col_id];
      }
      return max_row_offsets[col_id];
    },
    [column_categories = column_categories.begin(),
     is_non_list_parent,
     parent_col_ids = parent_col_ids.begin()] __device__(size_type col_id) {
      auto parent_col_id = parent_col_ids[col_id];
      // condition is true if parent is not a list, or sentinel/root
      return is_non_list_parent(parent_col_id);
    });

  // For Struct and List (to avoid copying entire strings when mixed type as string is enabled)
  thrust::transform_if(
    rmm::exec_policy(stream),
    col_range_begin.begin(),
    col_range_begin.end(),
    column_categories.begin(),
    col_range_end.begin(),
    [] __device__(auto i) { return i + 1; },
    [] __device__(NodeT type) { return type == NC_STRUCT || type == NC_LIST; });

  return std::tuple{tree_meta_t{std::move(column_categories),
                                std::move(parent_col_ids),
                                std::move(column_levels),
                                std::move(col_range_begin),
                                std::move(col_range_end)},
                    std::move(unique_col_ids),
                    std::move(max_row_offsets)};
}

}  // namespace detail
}  // namespace cudf::io::json
