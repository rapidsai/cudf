#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2024, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "nested_json.hpp"

#include <cudf/detail/nvtx/ranges.hpp>
#include <cudf/detail/utilities/vector_factories.hpp>
#include <cudf/types.hpp>
#include <cudf/utilities/error.hpp>
#include <cudf/utilities/memory_resource.hpp>
#include <cudf/utilities/span.hpp>

#include <rmm/cuda_stream_view.hpp>
#include <rmm/device_uvector.hpp>
#include <rmm/exec_policy.hpp>

#include <cuda/functional>
#include <thrust/for_each.h>
#include <thrust/iterator/constant_iterator.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/iterator/permutation_iterator.h>
#include <thrust/iterator/transform_output_iterator.h>
#include <thrust/iterator/zip_iterator.h>
#include <thrust/reduce.h>
#include <thrust/scan.h>
#include <thrust/sort.h>
#include <thrust/transform.h>
#include <thrust/transform_scan.h>
#include <thrust/unique.h>

namespace cudf::io::json {

using row_offset_t = size_type;

#ifdef CSR_DEBUG_PRINT
template <typename T>
void print(device_span<T const> d_vec, std::string name, rmm::cuda_stream_view stream)
{
  stream.synchronize();
  auto h_vec = cudf::detail::make_std_vector_sync(d_vec, stream);
  std::cout << name << " = ";
  for (auto e : h_vec) {
    std::cout << e << " ";
  }
  std::cout << std::endl;
}
#endif

namespace experimental::detail {

struct level_ordering {
  device_span<TreeDepthT const> node_levels;
  device_span<NodeIndexT const> col_ids;
  device_span<NodeIndexT const> parent_node_ids;
  __device__ bool operator()(NodeIndexT lhs_node_id, NodeIndexT rhs_node_id) const
  {
    auto lhs_parent_col_id = parent_node_ids[lhs_node_id] == parent_node_sentinel
                               ? parent_node_sentinel
                               : col_ids[parent_node_ids[lhs_node_id]];
    auto rhs_parent_col_id = parent_node_ids[rhs_node_id] == parent_node_sentinel
                               ? parent_node_sentinel
                               : col_ids[parent_node_ids[rhs_node_id]];

    return (node_levels[lhs_node_id] < node_levels[rhs_node_id]) ||
           (node_levels[lhs_node_id] == node_levels[rhs_node_id] &&
            lhs_parent_col_id < rhs_parent_col_id) ||
           (node_levels[lhs_node_id] == node_levels[rhs_node_id] &&
            lhs_parent_col_id == rhs_parent_col_id && col_ids[lhs_node_id] < col_ids[rhs_node_id]);
  }
};

struct parent_nodeids_to_colids {
  device_span<NodeIndexT const> rev_mapped_col_ids;
  __device__ auto operator()(NodeIndexT parent_node_id) -> NodeIndexT
  {
    return parent_node_id == parent_node_sentinel ? parent_node_sentinel
                                                  : rev_mapped_col_ids[parent_node_id];
  }
};

/**
 * @brief Reduces node tree representation to column tree CSR representation.
 *
 * @param node_tree Node tree representation of JSON string
 * @param original_col_ids Column ids of nodes
 * @param row_offsets Row offsets of nodes
 * @param is_array_of_arrays Whether the tree is an array of arrays
 * @param row_array_parent_col_id Column id of row array, if is_array_of_arrays is true
 * @param stream CUDA stream used for device memory operations and kernel launches
 * @return A tuple of column tree representation of JSON string, column ids of columns, and
 * max row offsets of columns
 */
std::tuple<compressed_sparse_row, column_tree_properties> reduce_to_column_tree(
  tree_meta_t& node_tree,
  device_span<NodeIndexT const> original_col_ids,
  device_span<NodeIndexT const> sorted_col_ids,
  device_span<NodeIndexT const> ordered_node_ids,
  device_span<row_offset_t const> row_offsets,
  bool is_array_of_arrays,
  NodeIndexT row_array_parent_col_id,
  rmm::cuda_stream_view stream)
{
  CUDF_FUNC_RANGE();

  if (original_col_ids.empty()) {
    rmm::device_uvector<NodeIndexT> empty_row_idx(0, stream);
    rmm::device_uvector<NodeIndexT> empty_col_idx(0, stream);
    rmm::device_uvector<NodeT> empty_column_categories(0, stream);
    rmm::device_uvector<row_offset_t> empty_max_row_offsets(0, stream);
    rmm::device_uvector<NodeIndexT> empty_mapped_col_ids(0, stream);
    return std::tuple{compressed_sparse_row{std::move(empty_row_idx), std::move(empty_col_idx)},
                      column_tree_properties{std::move(empty_column_categories),
                                             std::move(empty_max_row_offsets),
                                             std::move(empty_mapped_col_ids)}};
  }

  auto [unpermuted_tree, unpermuted_col_ids, unpermuted_max_row_offsets] =
    cudf::io::json::detail::reduce_to_column_tree(node_tree,
                                                  original_col_ids,
                                                  sorted_col_ids,
                                                  ordered_node_ids,
                                                  row_offsets,
                                                  is_array_of_arrays,
                                                  row_array_parent_col_id,
                                                  stream);

  NodeIndexT num_columns = unpermuted_col_ids.size();

  auto mapped_col_ids = cudf::detail::make_device_uvector_async(
    unpermuted_col_ids, stream, cudf::get_current_device_resource_ref());
  rmm::device_uvector<NodeIndexT> rev_mapped_col_ids(num_columns, stream);
  rmm::device_uvector<NodeIndexT> reordering_index(unpermuted_col_ids.size(), stream);

  thrust::sequence(
    rmm::exec_policy_nosync(stream), reordering_index.begin(), reordering_index.end());
  // Reorder nodes and column ids in level-wise fashion
  thrust::sort_by_key(
    rmm::exec_policy_nosync(stream),
    reordering_index.begin(),
    reordering_index.end(),
    mapped_col_ids.begin(),
    level_ordering{
      unpermuted_tree.node_levels, unpermuted_col_ids, unpermuted_tree.parent_node_ids});

  {
    auto mapped_col_ids_copy = cudf::detail::make_device_uvector_async(
      mapped_col_ids, stream, cudf::get_current_device_resource_ref());
    thrust::sequence(
      rmm::exec_policy_nosync(stream), rev_mapped_col_ids.begin(), rev_mapped_col_ids.end());
    thrust::sort_by_key(rmm::exec_policy_nosync(stream),
                        mapped_col_ids_copy.begin(),
                        mapped_col_ids_copy.end(),
                        rev_mapped_col_ids.begin());
  }

  rmm::device_uvector<NodeIndexT> parent_col_ids(num_columns, stream);
  thrust::transform_output_iterator parent_col_ids_it(parent_col_ids.begin(),
                                                      parent_nodeids_to_colids{rev_mapped_col_ids});
  rmm::device_uvector<row_offset_t> max_row_offsets(num_columns, stream);
  rmm::device_uvector<NodeT> column_categories(num_columns, stream);
  thrust::copy_n(
    rmm::exec_policy_nosync(stream),
    thrust::make_zip_iterator(thrust::make_permutation_iterator(
                                unpermuted_tree.parent_node_ids.begin(), reordering_index.begin()),
                              thrust::make_permutation_iterator(unpermuted_max_row_offsets.begin(),
                                                                reordering_index.begin()),
                              thrust::make_permutation_iterator(
                                unpermuted_tree.node_categories.begin(), reordering_index.begin())),
    num_columns,
    thrust::make_zip_iterator(
      parent_col_ids_it, max_row_offsets.begin(), column_categories.begin()));

#ifdef CSR_DEBUG_PRINT
  print<NodeIndexT>(reordering_index, "h_reordering_index", stream);
  print<NodeIndexT>(mapped_col_ids, "h_mapped_col_ids", stream);
  print<NodeIndexT>(rev_mapped_col_ids, "h_rev_mapped_col_ids", stream);
  print<NodeIndexT>(parent_col_ids, "h_parent_col_ids", stream);
  print<row_offset_t>(max_row_offsets, "h_max_row_offsets", stream);
#endif

  auto construct_row_idx = [&stream](NodeIndexT num_columns,
                                     device_span<NodeIndexT const> parent_col_ids) {
    auto row_idx = cudf::detail::make_zeroed_device_uvector_async<NodeIndexT>(
      static_cast<std::size_t>(num_columns + 1), stream, cudf::get_current_device_resource_ref());
    // Note that the first element of csr_parent_col_ids is -1 (parent_node_sentinel)
    // children adjacency

    auto num_non_leaf_columns = thrust::unique_count(
      rmm::exec_policy_nosync(stream), parent_col_ids.begin() + 1, parent_col_ids.end());
    rmm::device_uvector<NodeIndexT> non_leaf_nodes(num_non_leaf_columns, stream);
    rmm::device_uvector<NodeIndexT> non_leaf_nodes_children(num_non_leaf_columns, stream);
    thrust::reduce_by_key(rmm::exec_policy_nosync(stream),
                          parent_col_ids.begin() + 1,
                          parent_col_ids.end(),
                          thrust::make_constant_iterator(1),
                          non_leaf_nodes.begin(),
                          non_leaf_nodes_children.begin(),
                          thrust::equal_to<TreeDepthT>());

    thrust::scatter(rmm::exec_policy_nosync(stream),
                    non_leaf_nodes_children.begin(),
                    non_leaf_nodes_children.end(),
                    non_leaf_nodes.begin(),
                    row_idx.begin() + 1);

    if (num_columns > 1) {
      thrust::transform_inclusive_scan(
        rmm::exec_policy_nosync(stream),
        thrust::make_zip_iterator(thrust::make_counting_iterator(1), row_idx.begin() + 1),
        thrust::make_zip_iterator(thrust::make_counting_iterator(1) + num_columns, row_idx.end()),
        row_idx.begin() + 1,
        cuda::proclaim_return_type<NodeIndexT>([] __device__(auto a) {
          auto n   = thrust::get<0>(a);
          auto idx = thrust::get<1>(a);
          return n == 1 ? idx : idx + 1;
        }),
        thrust::plus<NodeIndexT>{});
    } else {
      auto single_node = 1;
      row_idx.set_element_async(1, single_node, stream);
    }

#ifdef CSR_DEBUG_PRINT
    print<NodeIndexT>(row_idx, "h_row_idx", stream);
#endif
    return row_idx;
  };

  auto construct_col_idx = [&stream](NodeIndexT num_columns,
                                     device_span<NodeIndexT const> parent_col_ids,
                                     device_span<NodeIndexT const> row_idx) {
    rmm::device_uvector<NodeIndexT> col_idx((num_columns - 1) * 2, stream);
    thrust::fill(rmm::exec_policy_nosync(stream), col_idx.begin(), col_idx.end(), -1);
    // excluding root node, construct scatter map
    rmm::device_uvector<NodeIndexT> map(num_columns - 1, stream);
    thrust::inclusive_scan_by_key(rmm::exec_policy_nosync(stream),
                                  parent_col_ids.begin() + 1,
                                  parent_col_ids.end(),
                                  thrust::make_constant_iterator(1),
                                  map.begin());
    thrust::for_each_n(rmm::exec_policy_nosync(stream),
                       thrust::make_counting_iterator(1),
                       num_columns - 1,
                       [row_idx        = row_idx.begin(),
                        map            = map.begin(),
                        parent_col_ids = parent_col_ids.begin()] __device__(auto i) {
                         auto parent_col_id = parent_col_ids[i];
                         if (parent_col_id == 0)
                           --map[i - 1];
                         else
                           map[i - 1] += row_idx[parent_col_id];
                       });
    thrust::scatter(rmm::exec_policy_nosync(stream),
                    thrust::make_counting_iterator(1),
                    thrust::make_counting_iterator(1) + num_columns - 1,
                    map.begin(),
                    col_idx.begin());

    // Skip the parent of root node
    thrust::scatter(rmm::exec_policy_nosync(stream),
                    parent_col_ids.begin() + 1,
                    parent_col_ids.end(),
                    row_idx.begin() + 1,
                    col_idx.begin());

#ifdef CSR_DEBUG_PRINT
    print<NodeIndexT>(col_idx, "h_col_idx", stream);
#endif

    return col_idx;
  };

  /*
    5. CSR construction:
      a. Sort column levels and get their ordering
      b. For each column node coln iterated according to sorted_column_levels; do
          i. Find nodes that have coln as the parent node -> set adj_coln
          ii. row idx[coln] = size of adj_coln + 1
          iii. col idx[coln] = adj_coln U {parent_col_id[coln]}
  */
  auto row_idx = construct_row_idx(num_columns, parent_col_ids);
  auto col_idx = construct_col_idx(num_columns, parent_col_ids, row_idx);

  return std::tuple{
    compressed_sparse_row{std::move(row_idx), std::move(col_idx)},
    column_tree_properties{
      std::move(column_categories), std::move(max_row_offsets), std::move(mapped_col_ids)}};
}

}  // namespace experimental::detail
}  // namespace cudf::io::json
