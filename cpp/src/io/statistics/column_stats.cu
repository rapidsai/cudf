#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019-2020, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "column_stats.h"

#include <io/utilities/block_utils.cuh>

#include <rmm/cuda_stream_view.hpp>

#include <hipcub/hipcub.hpp>

#include <hip/hip_math_constants.h>

constexpr int block_size = 1024;

namespace cudf {
namespace io {
/**
 * @brief shared state for statistics gather kernel
 */
struct stats_state_s {
  stats_column_desc col;                 ///< Column information
  statistics_group group;                ///< Group description
  statistics_chunk ck;                   ///< Output statistics chunk
  volatile statistics_val warp_min[32];  ///< Min reduction scratch
  volatile statistics_val warp_max[32];  ///< Max reduction scratch
};

/**
 * @brief shared state for statistics merge kernel
 */
struct merge_state_s {
  stats_column_desc col;                 ///< Column information
  statistics_merge_group group;          ///< Group description
  statistics_chunk ck;                   ///< Resulting statistics chunk
  volatile statistics_val warp_min[32];  ///< Min reduction scratch
  volatile statistics_val warp_max[32];  ///< Max reduction scratch
};

/**
 * Custom addition functor to ignore NaN inputs
 */
struct IgnoreNaNSum {
  __device__ __forceinline__ double operator()(const double &a, const double &b)
  {
    double aval = isnan(a) ? 0 : a;
    double bval = isnan(b) ? 0 : b;
    return aval + bval;
  }
};

/**
 * Warp-wide Min reduction for string types
 */
inline __device__ string_stats WarpReduceMinString(const char *smin, uint32_t lmin)
{
  uint32_t len = shuffle_xor(lmin, 1);
  const char *ptr =
    reinterpret_cast<const char *>(shuffle_xor(reinterpret_cast<uintptr_t>(smin), 1));
  if (!smin || (ptr && nvstr_is_lesser(ptr, len, smin, lmin))) {
    smin = ptr;
    lmin = len;
  }
  len = shuffle_xor(lmin, 2);
  ptr = reinterpret_cast<const char *>(shuffle_xor(reinterpret_cast<uintptr_t>(smin), 2));
  if (!smin || (ptr && nvstr_is_lesser(ptr, len, smin, lmin))) {
    smin = ptr;
    lmin = len;
  }
  len = shuffle_xor(lmin, 4);
  ptr = reinterpret_cast<const char *>(shuffle_xor(reinterpret_cast<uintptr_t>(smin), 4));
  if (!smin || (ptr && nvstr_is_lesser(ptr, len, smin, lmin))) {
    smin = ptr;
    lmin = len;
  }
  len = shuffle_xor(lmin, 8);
  ptr = reinterpret_cast<const char *>(shuffle_xor(reinterpret_cast<uintptr_t>(smin), 8));
  if (!smin || (ptr && nvstr_is_lesser(ptr, len, smin, lmin))) {
    smin = ptr;
    lmin = len;
  }
  len = shuffle_xor(lmin, 16);
  ptr = reinterpret_cast<const char *>(shuffle_xor(reinterpret_cast<uintptr_t>(smin), 16));
  if (!smin || (ptr && nvstr_is_lesser(ptr, len, smin, lmin))) {
    smin = ptr;
    lmin = len;
  }
  return {smin, lmin};
}

/**
 * Warp-wide Max reduction for string types
 */
inline __device__ string_stats WarpReduceMaxString(const char *smax, uint32_t lmax)
{
  uint32_t len = shuffle_xor(lmax, 1);
  const char *ptr =
    reinterpret_cast<const char *>(shuffle_xor(reinterpret_cast<uintptr_t>(smax), 1));
  if (!smax || (ptr && nvstr_is_greater(ptr, len, smax, lmax))) {
    smax = ptr;
    lmax = len;
  }
  len = shuffle_xor(lmax, 2);
  ptr = reinterpret_cast<const char *>(shuffle_xor(reinterpret_cast<uintptr_t>(smax), 2));
  if (!smax || (ptr && nvstr_is_greater(ptr, len, smax, lmax))) {
    smax = ptr;
    lmax = len;
  }
  len = shuffle_xor(lmax, 4);
  ptr = reinterpret_cast<const char *>(shuffle_xor(reinterpret_cast<uintptr_t>(smax), 4));
  if (!smax || (ptr && nvstr_is_greater(ptr, len, smax, lmax))) {
    smax = ptr;
    lmax = len;
  }
  len = shuffle_xor(lmax, 8);
  ptr = reinterpret_cast<const char *>(shuffle_xor(reinterpret_cast<uintptr_t>(smax), 8));
  if (!smax || (ptr && nvstr_is_greater(ptr, len, smax, lmax))) {
    smax = ptr;
    lmax = len;
  }
  len = shuffle_xor(lmax, 16);
  ptr = reinterpret_cast<const char *>(shuffle_xor(reinterpret_cast<uintptr_t>(smax), 16));
  if (!smax || (ptr && nvstr_is_greater(ptr, len, smax, lmax))) {
    smax = ptr;
    lmax = len;
  }
  return {smax, lmax};
}

/**
 * @brief Gather statistics for integer-like columns
 *
 * @param s shared block state
 * @param dtype data type
 * @param t thread id
 * @param storage temporary storage for reduction
 */
template <typename Storage>
void __device__
gatherIntColumnStats(stats_state_s *s, statistics_dtype dtype, uint32_t t, Storage &storage)
{
  using block_reduce = hipcub::BlockReduce<int64_t, block_size>;
  int64_t vmin       = INT64_MAX;
  int64_t vmax       = INT64_MIN;
  int64_t vsum       = 0;
  int64_t v;
  uint32_t nn_cnt = 0;
  __shared__ volatile bool has_minmax;
  for (uint32_t i = 0; i < s->group.num_rows; i += block_size) {
    uint32_t r                = i + t;
    uint32_t row              = r + s->group.start_row;
    const uint32_t *valid_map = s->col.valid_map_base;
    uint32_t is_valid         = (r < s->group.num_rows && row < s->col.num_values)
                          ? (valid_map) ? (valid_map[(row + s->col.column_offset) / 32] >>
                                           ((row + s->col.column_offset) % 32)) &
                                            1
                                        : 1
                          : 0;
    if (is_valid) {
      switch (dtype) {
        case dtype_int32:
        case dtype_date32: v = static_cast<const int32_t *>(s->col.column_data_base)[row]; break;
        case dtype_int64:
        case dtype_decimal64: v = static_cast<const int64_t *>(s->col.column_data_base)[row]; break;
        case dtype_int16: v = static_cast<const int16_t *>(s->col.column_data_base)[row]; break;
        case dtype_timestamp64:
          v = static_cast<const int64_t *>(s->col.column_data_base)[row];
          if (s->col.ts_scale < -1) {
            v /= -s->col.ts_scale;
          } else if (s->col.ts_scale > 1) {
            v *= s->col.ts_scale;
          }
          break;
        default: v = static_cast<const int8_t *>(s->col.column_data_base)[row]; break;
      }
      vmin = min(vmin, v);
      vmax = max(vmax, v);
      vsum += v;
    }
    nn_cnt += __syncthreads_count(is_valid);
  }
  if (!t) {
    s->ck.non_nulls  = nn_cnt;
    s->ck.null_count = s->group.num_rows - nn_cnt;
  }
  vmin = block_reduce(storage.integer_stats).Reduce(vmin, hipcub::Min());
  __syncthreads();
  vmax = block_reduce(storage.integer_stats).Reduce(vmax, hipcub::Max());
  if (!t) { has_minmax = (vmin <= vmax); }
  __syncthreads();
  if (has_minmax) { vsum = block_reduce(storage.integer_stats).Sum(vsum); }
  if (!t) {
    if (has_minmax) {
      s->ck.min_value.i_val = vmin;
      s->ck.max_value.i_val = vmax;
      s->ck.sum.i_val       = vsum;
    }
    s->ck.has_minmax = has_minmax;
    // TODO: For now, don't set the sum flag with 64-bit values so we don't have to check for
    // 64-bit sum overflow
    s->ck.has_sum = (dtype <= dtype_int32 && has_minmax);
  }
}

/**
 * @brief Gather statistics for floating-point columns
 *
 * @param s shared block state
 * @param dtype data type
 * @param t thread id
 * @param storage temporary storage for reduction
 */
template <typename Storage>
void __device__
gatherFloatColumnStats(stats_state_s *s, statistics_dtype dtype, uint32_t t, Storage &storage)
{
  using block_reduce = hipcub::BlockReduce<double, block_size>;
  double vmin        = HIP_INF;
  double vmax        = -HIP_INF;
  double vsum        = 0;
  double v;
  uint32_t nn_cnt = 0;
  __shared__ volatile bool has_minmax;
  for (uint32_t i = 0; i < s->group.num_rows; i += block_size) {
    uint32_t r                = i + t;
    uint32_t row              = r + s->group.start_row;
    const uint32_t *valid_map = s->col.valid_map_base;
    uint32_t is_valid         = (r < s->group.num_rows && row < s->col.num_values)
                          ? (valid_map) ? (valid_map[(row + s->col.column_offset) >> 5] >>
                                           ((row + s->col.column_offset) & 0x1f)) &
                                            1
                                        : 1
                          : 0;
    if (is_valid) {
      if (dtype == dtype_float64) {
        v = static_cast<const double *>(s->col.column_data_base)[row];
      } else {
        v = static_cast<const float *>(s->col.column_data_base)[row];
      }
      if (v < vmin) { vmin = v; }
      if (v > vmax) { vmax = v; }
      if (!isnan(v)) { vsum += v; }
    }
    nn_cnt += __syncthreads_count(is_valid);
  }
  if (!t) {
    s->ck.non_nulls  = nn_cnt;
    s->ck.null_count = s->group.num_rows - nn_cnt;
  }
  vmin = block_reduce(storage.float_stats).Reduce(vmin, hipcub::Min());
  __syncthreads();
  vmax = block_reduce(storage.float_stats).Reduce(vmax, hipcub::Max());
  if (!t) { has_minmax = (vmin <= vmax); }
  __syncthreads();
  if (has_minmax) { vsum = block_reduce(storage.float_stats).Reduce(vsum, IgnoreNaNSum()); }
  if (!t) {
    if (has_minmax) {
      s->ck.min_value.fp_val = (vmin != 0.0) ? vmin : HIP_NEG_ZERO;
      s->ck.max_value.fp_val = (vmax != 0.0) ? vmax : HIP_ZERO;
      s->ck.sum.fp_val       = vsum;
    }
    s->ck.has_minmax = has_minmax;
    s->ck.has_sum    = has_minmax;  // Implies sum is valid as well
  }
}

// FIXME: Use native libcudf string type
struct nvstrdesc_s {
  const char *ptr;
  size_t count;
};

/**
 * @brief Gather statistics for string columns
 *
 * @param s shared block state
 * @param t thread id
 * @param storage temporary storage for reduction
 */
template <typename Storage>
void __device__ gatherStringColumnStats(stats_state_s *s, uint32_t t, Storage &storage)
{
  using block_reduce = hipcub::BlockReduce<uint32_t, block_size>;
  uint32_t len_sum   = 0;
  const char *smin   = nullptr;
  const char *smax   = nullptr;
  uint32_t lmin      = 0;
  uint32_t lmax      = 0;
  uint32_t nn_cnt    = 0;
  bool has_minmax;
  string_stats minval, maxval;

  for (uint32_t i = 0; i < s->group.num_rows; i += block_size) {
    uint32_t r                = i + t;
    uint32_t row              = r + s->group.start_row;
    const uint32_t *valid_map = s->col.valid_map_base;
    uint32_t is_valid         = (r < s->group.num_rows && row < s->col.num_values)
                          ? (valid_map) ? (valid_map[(row + s->col.column_offset) >> 5] >>
                                           ((row + s->col.column_offset) & 0x1f)) &
                                            1
                                        : 1
                          : 0;
    if (is_valid) {
      const nvstrdesc_s *str_col = static_cast<const nvstrdesc_s *>(s->col.column_data_base);
      uint32_t len               = (uint32_t)str_col[row].count;
      const char *ptr            = str_col[row].ptr;
      len_sum += len;
      if (!smin || nvstr_is_lesser(ptr, len, smin, lmin)) {
        lmin = len;
        smin = ptr;
      }
      if (!smax || nvstr_is_greater(ptr, len, smax, lmax)) {
        lmax = len;
        smax = ptr;
      }
    }
    nn_cnt += __syncthreads_count(is_valid);
  }
  if (!t) {
    s->ck.non_nulls  = nn_cnt;
    s->ck.null_count = s->group.num_rows - nn_cnt;
  }
  minval = WarpReduceMinString(smin, lmin);
  maxval = WarpReduceMaxString(smax, lmax);
  __syncwarp();
  if (!(t & 0x1f)) {
    s->warp_min[t >> 5].str_val.ptr    = minval.ptr;
    s->warp_min[t >> 5].str_val.length = minval.length;
    s->warp_max[t >> 5].str_val.ptr    = maxval.ptr;
    s->warp_max[t >> 5].str_val.length = maxval.length;
  }
  has_minmax = __syncthreads_or(smin != nullptr);
  if (has_minmax) { len_sum = block_reduce(storage.string_stats).Sum(len_sum); }
  if (t < 32 * 1) {
    minval = WarpReduceMinString(s->warp_min[t].str_val.ptr, s->warp_min[t].str_val.length);
    if (!(t & 0x1f)) {
      if (has_minmax) {
        s->ck.min_value.str_val.ptr    = minval.ptr;
        s->ck.min_value.str_val.length = minval.length;
        s->ck.sum.i_val                = len_sum;
      }
      s->ck.has_minmax = has_minmax;
      s->ck.has_sum    = has_minmax;
    }
  } else if (t < 32 * 2 and has_minmax) {
    maxval =
      WarpReduceMaxString(s->warp_max[t & 0x1f].str_val.ptr, s->warp_max[t & 0x1f].str_val.length);
    if (!(t & 0x1f)) {
      s->ck.max_value.str_val.ptr    = maxval.ptr;
      s->ck.max_value.str_val.length = maxval.length;
    }
  }
}

/**
 * @brief Gather column chunk statistics (min/max values, sum and null count)
 * for a group of rows.
 *
 * blockDim {1024,1,1}
 *
 * @param chunks Destination statistics results
 * @param groups Statistics source information
 */
template <int block_size>
__global__ void __launch_bounds__(block_size, 1)
  gpuGatherColumnStatistics(statistics_chunk *chunks, const statistics_group *groups)
{
  __shared__ __align__(8) stats_state_s state_g;
  __shared__ union {
    typename hipcub::BlockReduce<int64_t, block_size>::TempStorage integer_stats;
    typename hipcub::BlockReduce<double, block_size>::TempStorage float_stats;
    typename hipcub::BlockReduce<uint32_t, block_size>::TempStorage string_stats;
  } temp_storage;

  stats_state_s *const s = &state_g;
  uint32_t t             = threadIdx.x;
  statistics_dtype dtype;

  if (t < sizeof(statistics_group) / sizeof(uint32_t)) {
    reinterpret_cast<uint32_t *>(&s->group)[t] =
      reinterpret_cast<const uint32_t *>(&groups[blockIdx.x])[t];
  }
  if (t < sizeof(statistics_chunk) / sizeof(uint32_t)) {
    reinterpret_cast<uint32_t *>(&s->ck)[t] = 0;
  }
  __syncthreads();
  if (t < sizeof(stats_column_desc) / sizeof(uint32_t)) {
    reinterpret_cast<uint32_t *>(&s->col)[t] = reinterpret_cast<const uint32_t *>(s->group.col)[t];
  }
  __syncthreads();
  dtype = s->col.stats_dtype;
  if (dtype >= dtype_bool && dtype <= dtype_decimal64) {
    gatherIntColumnStats(s, dtype, t, temp_storage);
  } else if (dtype >= dtype_float32 && dtype <= dtype_float64) {
    gatherFloatColumnStats(s, dtype, t, temp_storage);
  } else if (dtype == dtype_string) {
    gatherStringColumnStats(s, t, temp_storage);
  }
  __syncthreads();
  if (t < sizeof(statistics_chunk) / sizeof(uint32_t)) {
    reinterpret_cast<uint32_t *>(&chunks[blockIdx.x])[t] = reinterpret_cast<uint32_t *>(&s->ck)[t];
  }
}

/**
 * @brief Merge statistics for integer-like columns
 *
 * @param s shared block state
 * @param dtype data type
 * @param ck_in pointer to first statistic chunk
 * @param num_chunks number of statistic chunks to merge
 * @param t thread id
 * @param storage temporary storage for reduction
 */
template <typename Storage>
void __device__ mergeIntColumnStats(merge_state_s *s,
                                    statistics_dtype dtype,
                                    const statistics_chunk *ck_in,
                                    uint32_t num_chunks,
                                    uint32_t t,
                                    Storage &storage)
{
  int64_t vmin        = INT64_MAX;
  int64_t vmax        = INT64_MIN;
  int64_t vsum        = 0;
  uint32_t non_nulls  = 0;
  uint32_t null_count = 0;
  __shared__ volatile bool has_minmax;
  for (uint32_t i = t; i < num_chunks; i += block_size) {
    const statistics_chunk *ck = &ck_in[i];
    if (ck->has_minmax) {
      vmin = min(vmin, ck->min_value.i_val);
      vmax = max(vmax, ck->max_value.i_val);
    }
    if (ck->has_sum) { vsum += ck->sum.i_val; }
    non_nulls += ck->non_nulls;
    null_count += ck->null_count;
  }
  vmin = hipcub::BlockReduce<int64_t, block_size>(storage.i64).Reduce(vmin, hipcub::Min());
  __syncthreads();
  vmax = hipcub::BlockReduce<int64_t, block_size>(storage.i64).Reduce(vmax, hipcub::Max());
  if (!t) { has_minmax = (vmin <= vmax); }
  __syncthreads();
  non_nulls = hipcub::BlockReduce<uint32_t, block_size>(storage.u32).Sum(non_nulls);
  __syncthreads();
  null_count = hipcub::BlockReduce<uint32_t, block_size>(storage.u32).Sum(null_count);
  __syncthreads();
  if (has_minmax) { vsum = hipcub::BlockReduce<int64_t, block_size>(storage.i64).Sum(vsum); }

  if (!t) {
    if (has_minmax) {
      s->ck.min_value.i_val = vmin;
      s->ck.max_value.i_val = vmax;
      s->ck.sum.i_val       = vsum;
    }
    s->ck.has_minmax = has_minmax;
    // TODO: For now, don't set the sum flag with 64-bit values so we don't have to check for
    // 64-bit sum overflow
    s->ck.has_sum    = (dtype <= dtype_int32 && has_minmax);
    s->ck.non_nulls  = non_nulls;
    s->ck.null_count = null_count;
  }
}

/**
 * @brief Merge statistics for floating-point columns
 *
 * @param s shared block state
 * @param dtype data type
 * @param ck_in pointer to first statistic chunk
 * @param num_chunks number of statistic chunks to merge
 * @param t thread id
 * @param storage temporary storage for reduction
 */
template <typename Storage>
void __device__ mergeFloatColumnStats(merge_state_s *s,
                                      const statistics_chunk *ck_in,
                                      uint32_t num_chunks,
                                      uint32_t t,
                                      Storage &storage)
{
  double vmin         = HIP_INF;
  double vmax         = -HIP_INF;
  double vsum         = 0;
  uint32_t non_nulls  = 0;
  uint32_t null_count = 0;
  __shared__ volatile bool has_minmax;
  for (uint32_t i = t; i < num_chunks; i += block_size) {
    const statistics_chunk *ck = &ck_in[i];
    if (ck->has_minmax) {
      double v0 = ck->min_value.fp_val;
      double v1 = ck->max_value.fp_val;
      if (v0 < vmin) { vmin = v0; }
      if (v1 > vmax) { vmax = v1; }
    }
    if (ck->has_sum) { vsum += ck->sum.fp_val; }
    non_nulls += ck->non_nulls;
    null_count += ck->null_count;
  }

  vmin = hipcub::BlockReduce<double, block_size>(storage.f64).Reduce(vmin, hipcub::Min());
  __syncthreads();
  vmax = hipcub::BlockReduce<double, block_size>(storage.f64).Reduce(vmax, hipcub::Max());
  if (!t) { has_minmax = (vmin <= vmax); }
  __syncthreads();
  non_nulls = hipcub::BlockReduce<uint32_t, block_size>(storage.u32).Sum(non_nulls);
  __syncthreads();
  null_count = hipcub::BlockReduce<uint32_t, block_size>(storage.u32).Sum(null_count);
  __syncthreads();
  if (has_minmax) {
    vsum = hipcub::BlockReduce<double, block_size>(storage.f64).Reduce(vsum, IgnoreNaNSum());
  }

  if (!t) {
    if (has_minmax) {
      s->ck.min_value.fp_val = (vmin != 0.0) ? vmin : HIP_NEG_ZERO;
      s->ck.max_value.fp_val = (vmax != 0.0) ? vmax : HIP_ZERO;
      s->ck.sum.fp_val       = vsum;
    }
    s->ck.has_minmax = has_minmax;
    s->ck.has_sum    = has_minmax;  // Implies sum is valid as well
    s->ck.non_nulls  = non_nulls;
    s->ck.null_count = null_count;
  }
}

/**
 * @brief Merge statistics for string columns
 *
 * @param s shared block state
 * @param ck_in pointer to first statistic chunk
 * @param num_chunks number of statistic chunks to merge
 * @param t thread id
 * @param storage temporary storage for reduction
 */
template <typename Storage>
void __device__ mergeStringColumnStats(merge_state_s *s,
                                       const statistics_chunk *ck_in,
                                       uint32_t num_chunks,
                                       uint32_t t,
                                       Storage &storage)
{
  uint32_t len_sum    = 0;
  const char *smin    = nullptr;
  const char *smax    = nullptr;
  uint32_t lmin       = 0;
  uint32_t lmax       = 0;
  uint32_t non_nulls  = 0;
  uint32_t null_count = 0;
  bool has_minmax;
  string_stats minval, maxval;

  for (uint32_t i = t; i < num_chunks; i += block_size) {
    const statistics_chunk *ck = &ck_in[i];
    if (ck->has_minmax) {
      uint32_t len0    = ck->min_value.str_val.length;
      const char *ptr0 = ck->min_value.str_val.ptr;
      uint32_t len1    = ck->max_value.str_val.length;
      const char *ptr1 = ck->max_value.str_val.ptr;
      if (!smin || (ptr0 && nvstr_is_lesser(ptr0, len0, smin, lmin))) {
        lmin = len0;
        smin = ptr0;
      }
      if (!smax || (ptr1 && nvstr_is_greater(ptr1, len1, smax, lmax))) {
        lmax = len1;
        smax = ptr1;
      }
    }
    if (ck->has_sum) { len_sum += (uint32_t)ck->sum.i_val; }
    non_nulls += ck->non_nulls;
    null_count += ck->null_count;
  }
  minval = WarpReduceMinString(smin, lmin);
  maxval = WarpReduceMaxString(smax, lmax);
  if (!(t & 0x1f)) {
    s->warp_min[t >> 5].str_val.ptr    = minval.ptr;
    s->warp_min[t >> 5].str_val.length = minval.length;
    s->warp_max[t >> 5].str_val.ptr    = maxval.ptr;
    s->warp_max[t >> 5].str_val.length = maxval.length;
  }
  has_minmax = __syncthreads_or(smin != nullptr);

  non_nulls = hipcub::BlockReduce<uint32_t, block_size>(storage.u32).Sum(non_nulls);
  __syncthreads();
  null_count = hipcub::BlockReduce<uint32_t, block_size>(storage.u32).Sum(null_count);
  __syncthreads();
  if (has_minmax) { len_sum = hipcub::BlockReduce<uint32_t, block_size>(storage.u32).Sum(len_sum); }
  if (t < 32 * 1) {
    minval = WarpReduceMinString(s->warp_min[t].str_val.ptr, s->warp_min[t].str_val.length);
    if (!(t & 0x1f)) {
      if (has_minmax) {
        s->ck.min_value.str_val.ptr    = minval.ptr;
        s->ck.min_value.str_val.length = minval.length;
        s->ck.sum.i_val                = len_sum;
      }
      s->ck.has_minmax = has_minmax;
      s->ck.has_sum    = has_minmax;
      s->ck.non_nulls  = non_nulls;
      s->ck.null_count = null_count;
    }
  } else if (t < 32 * 2) {
    maxval =
      WarpReduceMaxString(s->warp_max[t & 0x1f].str_val.ptr, s->warp_max[t & 0x1f].str_val.length);
    if (!((t & 0x1f) and has_minmax)) {
      s->ck.max_value.str_val.ptr    = maxval.ptr;
      s->ck.max_value.str_val.length = maxval.length;
    }
  }
}

/**
 * @brief Combine multiple statistics chunk together to form new statistics chunks
 *
 * blockDim {1024,1,1}
 *
 * @param chunks_out Destination statistic chunks
 * @param chunks_in Source statistic chunks
 * @param groups Statistic chunk grouping information
 */
template <int block_size>
__global__ void __launch_bounds__(block_size, 1)
  gpuMergeColumnStatistics(statistics_chunk *chunks_out,
                           const statistics_chunk *chunks_in,
                           const statistics_merge_group *groups)
{
  __shared__ __align__(8) merge_state_s state_g;
  __shared__ struct {
    typename hipcub::BlockReduce<uint32_t, block_size>::TempStorage u32;
    typename hipcub::BlockReduce<int64_t, block_size>::TempStorage i64;
    typename hipcub::BlockReduce<double, block_size>::TempStorage f64;
  } storage;

  merge_state_s *const s = &state_g;
  uint32_t t             = threadIdx.x;
  statistics_dtype dtype;

  if (t < sizeof(statistics_merge_group) / sizeof(uint32_t)) {
    reinterpret_cast<uint32_t *>(&s->group)[t] =
      reinterpret_cast<const uint32_t *>(&groups[blockIdx.x])[t];
  }
  __syncthreads();
  if (t < sizeof(stats_column_desc) / sizeof(uint32_t)) {
    reinterpret_cast<uint32_t *>(&s->col)[t] = reinterpret_cast<const uint32_t *>(s->group.col)[t];
  }
  __syncthreads();
  dtype = s->col.stats_dtype;

  if (dtype >= dtype_bool && dtype <= dtype_decimal64) {
    mergeIntColumnStats(
      s, dtype, chunks_in + s->group.start_chunk, s->group.num_chunks, t, storage);
  } else if (dtype >= dtype_float32 && dtype <= dtype_float64) {
    mergeFloatColumnStats(s, chunks_in + s->group.start_chunk, s->group.num_chunks, t, storage);
  } else if (dtype == dtype_string) {
    mergeStringColumnStats(s, chunks_in + s->group.start_chunk, s->group.num_chunks, t, storage);
  }

  __syncthreads();
  if (t < sizeof(statistics_chunk) / sizeof(uint32_t)) {
    reinterpret_cast<uint32_t *>(&chunks_out[blockIdx.x])[t] =
      reinterpret_cast<uint32_t *>(&s->ck)[t];
  }
}

/**
 * @brief Launches kernel to gather column statistics
 *
 * @param[out] chunks Statistics results [num_chunks]
 * @param[in] groups Statistics row groups [num_chunks]
 * @param[in] num_chunks Number of chunks & rowgroups
 * @param[in] stream CUDA stream to use, default 0
 */
void GatherColumnStatistics(statistics_chunk *chunks,
                            const statistics_group *groups,
                            uint32_t num_chunks,
                            rmm::cuda_stream_view stream)
{
  gpuGatherColumnStatistics<block_size>
    <<<num_chunks, block_size, 0, stream.value()>>>(chunks, groups);
}

/**
 * @brief Launches kernel to merge column statistics
 *
 * @param[out] chunks_out Statistics results [num_chunks]
 * @param[out] chunks_in Input statistics
 * @param[in] groups Statistics groups [num_chunks]
 * @param[in] num_chunks Number of chunks & groups
 * @param[in] stream CUDA stream to use, default 0
 */
void MergeColumnStatistics(statistics_chunk *chunks_out,
                           const statistics_chunk *chunks_in,
                           const statistics_merge_group *groups,
                           uint32_t num_chunks,
                           rmm::cuda_stream_view stream)
{
  gpuMergeColumnStatistics<block_size>
    <<<num_chunks, block_size, 0, stream.value()>>>(chunks_out, chunks_in, groups);
}

}  // namespace io
}  // namespace cudf
