#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019-2020, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "column_stats.h"

#include <io/utilities/block_utils.cuh>

#include <rmm/cuda_stream_view.hpp>

#include <hipcub/hipcub.hpp>

#include <hip/hip_math_constants.h>

namespace cudf {
namespace io {
/**
 * @brief shared state for statistics gather kernel
 */
struct stats_state_s {
  stats_column_desc col;                 ///< Column information
  statistics_group group;                ///< Group description
  statistics_chunk ck;                   ///< Output statistics chunk
  volatile statistics_val warp_min[32];  ///< Min reduction scratch
  volatile statistics_val warp_max[32];  ///< Max reduction scratch
  volatile statistics_val warp_sum[32];  ///< Sum reduction scratch
};

/**
 * @brief shared state for statistics merge kernel
 */
struct merge_state_s {
  stats_column_desc col;                 ///< Column information
  statistics_merge_group group;          ///< Group description
  statistics_chunk ck;                   ///< Resulting statistics chunk
  volatile statistics_val warp_min[32];  ///< Min reduction scratch
  volatile statistics_val warp_max[32];  ///< Max reduction scratch
  volatile statistics_val warp_sum[32];  ///< Sum reduction scratch
  volatile uint32_t warp_non_nulls[32];  ///< Non-nulls reduction scratch
  volatile uint32_t warp_nulls[32];      ///< Nulls reduction scratch
};

/**
 * Custom addition functor to ignore NaN inputs
 */
struct IgnoreNaNSum {
  __device__ __forceinline__ double operator()(const double &a, const double &b)
  {
    double aval = isnan(a) ? 0 : a;
    double bval = isnan(b) ? 0 : b;
    return aval + bval;
  }
};

/**
 * Warp-wide Min reduction for string types
 */
inline __device__ string_stats WarpReduceMinString(const char *smin, uint32_t lmin)
{
  uint32_t len = shuffle_xor(lmin, 1);
  const char *ptr =
    reinterpret_cast<const char *>(shuffle_xor(reinterpret_cast<uintptr_t>(smin), 1));
  if (!smin || (ptr && nvstr_is_lesser(ptr, len, smin, lmin))) {
    smin = ptr;
    lmin = len;
  }
  len = shuffle_xor(lmin, 2);
  ptr = reinterpret_cast<const char *>(shuffle_xor(reinterpret_cast<uintptr_t>(smin), 2));
  if (!smin || (ptr && nvstr_is_lesser(ptr, len, smin, lmin))) {
    smin = ptr;
    lmin = len;
  }
  len = shuffle_xor(lmin, 4);
  ptr = reinterpret_cast<const char *>(shuffle_xor(reinterpret_cast<uintptr_t>(smin), 4));
  if (!smin || (ptr && nvstr_is_lesser(ptr, len, smin, lmin))) {
    smin = ptr;
    lmin = len;
  }
  len = shuffle_xor(lmin, 8);
  ptr = reinterpret_cast<const char *>(shuffle_xor(reinterpret_cast<uintptr_t>(smin), 8));
  if (!smin || (ptr && nvstr_is_lesser(ptr, len, smin, lmin))) {
    smin = ptr;
    lmin = len;
  }
  len = shuffle_xor(lmin, 16);
  ptr = reinterpret_cast<const char *>(shuffle_xor(reinterpret_cast<uintptr_t>(smin), 16));
  if (!smin || (ptr && nvstr_is_lesser(ptr, len, smin, lmin))) {
    smin = ptr;
    lmin = len;
  }
  return {smin, lmin};
}

/**
 * Warp-wide Max reduction for string types
 */
inline __device__ string_stats WarpReduceMaxString(const char *smax, uint32_t lmax)
{
  uint32_t len = shuffle_xor(lmax, 1);
  const char *ptr =
    reinterpret_cast<const char *>(shuffle_xor(reinterpret_cast<uintptr_t>(smax), 1));
  if (!smax || (ptr && nvstr_is_greater(ptr, len, smax, lmax))) {
    smax = ptr;
    lmax = len;
  }
  len = shuffle_xor(lmax, 2);
  ptr = reinterpret_cast<const char *>(shuffle_xor(reinterpret_cast<uintptr_t>(smax), 2));
  if (!smax || (ptr && nvstr_is_greater(ptr, len, smax, lmax))) {
    smax = ptr;
    lmax = len;
  }
  len = shuffle_xor(lmax, 4);
  ptr = reinterpret_cast<const char *>(shuffle_xor(reinterpret_cast<uintptr_t>(smax), 4));
  if (!smax || (ptr && nvstr_is_greater(ptr, len, smax, lmax))) {
    smax = ptr;
    lmax = len;
  }
  len = shuffle_xor(lmax, 8);
  ptr = reinterpret_cast<const char *>(shuffle_xor(reinterpret_cast<uintptr_t>(smax), 8));
  if (!smax || (ptr && nvstr_is_greater(ptr, len, smax, lmax))) {
    smax = ptr;
    lmax = len;
  }
  len = shuffle_xor(lmax, 16);
  ptr = reinterpret_cast<const char *>(shuffle_xor(reinterpret_cast<uintptr_t>(smax), 16));
  if (!smax || (ptr && nvstr_is_greater(ptr, len, smax, lmax))) {
    smax = ptr;
    lmax = len;
  }
  return {smax, lmax};
}

/**
 * @brief Gather statistics for integer-like columns
 *
 * @param s shared block state
 * @param dtype data type
 * @param t thread id
 * @param storage temporary storage for warp reduction
 */
template <typename Storage>
void __device__
gatherIntColumnStats(stats_state_s *s, statistics_dtype dtype, uint32_t t, Storage &storage)
{
  using warp_reduce = hipcub::WarpReduce<int64_t>;
  int64_t vmin      = INT64_MAX;
  int64_t vmax      = INT64_MIN;
  int64_t vsum      = 0;
  int64_t v;
  uint32_t nn_cnt = 0;
  bool has_minmax;
  for (uint32_t i = 0; i < s->group.num_rows; i += 1024) {
    uint32_t r                = i + t;
    uint32_t row              = r + s->group.start_row;
    const uint32_t *valid_map = s->col.valid_map_base;
    uint32_t is_valid         = (r < s->group.num_rows && row < s->col.num_values)
                          ? (valid_map) ? (valid_map[(row + s->col.column_offset) / 32] >>
                                           ((row + s->col.column_offset) % 32)) &
                                            1
                                        : 1
                          : 0;
    if (is_valid) {
      switch (dtype) {
        case dtype_int32:
        case dtype_date32: v = static_cast<const int32_t *>(s->col.column_data_base)[row]; break;
        case dtype_int64:
        case dtype_decimal64: v = static_cast<const int64_t *>(s->col.column_data_base)[row]; break;
        case dtype_int16: v = static_cast<const int16_t *>(s->col.column_data_base)[row]; break;
        case dtype_timestamp64:
          v = static_cast<const int64_t *>(s->col.column_data_base)[row];
          if (s->col.ts_scale < -1) {
            v /= -s->col.ts_scale;
          } else if (s->col.ts_scale > 1) {
            v *= s->col.ts_scale;
          }
          break;
        default: v = static_cast<const int8_t *>(s->col.column_data_base)[row]; break;
      }
      vmin = min(vmin, v);
      vmax = max(vmax, v);
      vsum += v;
    }
    nn_cnt += __syncthreads_count(is_valid);
  }
  if (!t) {
    s->ck.non_nulls  = nn_cnt;
    s->ck.null_count = s->group.num_rows - nn_cnt;
  }
  vmin = warp_reduce(storage.integer_stats[t / 32]).Reduce(vmin, hipcub::Min());
  vmin = shuffle(vmin);
  vmax = warp_reduce(storage.integer_stats[t / 32]).Reduce(vmax, hipcub::Max());
  vmax = shuffle(vmax);
  vsum = warp_reduce(storage.integer_stats[t / 32]).Sum(vsum);
  if (!(t & 0x1f)) {
    s->warp_min[t >> 5].i_val = vmin;
    s->warp_max[t >> 5].i_val = vmax;
    s->warp_sum[t >> 5].i_val = vsum;
  }
  has_minmax = __syncthreads_or(vmin <= vmax);
  if (t < 32 * 1) {
    vmin = warp_reduce(storage.integer_stats[t / 32]).Reduce(s->warp_min[t].i_val, hipcub::Min());
    if (!(t & 0x1f)) {
      s->ck.min_value.i_val = vmin;
      s->ck.has_minmax      = (has_minmax);
    }
  } else if (t < 32 * 2) {
    vmax =
      warp_reduce(storage.integer_stats[t / 32]).Reduce(s->warp_max[t & 0x1f].i_val, hipcub::Max());
    if (!(t & 0x1f)) { s->ck.max_value.i_val = vmax; }
  } else if (t < 32 * 3) {
    vsum = warp_reduce(storage.integer_stats[t / 32]).Sum(s->warp_sum[t & 0x1f].i_val);
    if (!(t & 0x1f)) {
      s->ck.sum.i_val = vsum;
      // TODO: For now, don't set the sum flag with 64-bit values so we don't have to check for
      // 64-bit sum overflow
      s->ck.has_sum = (dtype <= dtype_int32 && has_minmax);
    }
  }
}

/**
 * @brief Gather statistics for floating-point columns
 *
 * @param s shared block state
 * @param dtype data type
 * @param t thread id
 * @param storage temporary storage for warp reduction
 */
template <typename Storage>
void __device__
gatherFloatColumnStats(stats_state_s *s, statistics_dtype dtype, uint32_t t, Storage &storage)
{
  using warp_reduce = hipcub::WarpReduce<double>;
  double vmin       = HIP_INF;
  double vmax       = -HIP_INF;
  double vsum       = 0;
  double v;
  uint32_t nn_cnt = 0;
  bool has_minmax;
  for (uint32_t i = 0; i < s->group.num_rows; i += 1024) {
    uint32_t r                = i + t;
    uint32_t row              = r + s->group.start_row;
    const uint32_t *valid_map = s->col.valid_map_base;
    uint32_t is_valid         = (r < s->group.num_rows && row < s->col.num_values)
                          ? (valid_map) ? (valid_map[(row + s->col.column_offset) >> 5] >>
                                           ((row + s->col.column_offset) & 0x1f)) &
                                            1
                                        : 1
                          : 0;
    if (is_valid) {
      if (dtype == dtype_float64) {
        v = static_cast<const double *>(s->col.column_data_base)[row];
      } else {
        v = static_cast<const float *>(s->col.column_data_base)[row];
      }
      if (v < vmin) { vmin = v; }
      if (v > vmax) { vmax = v; }
      if (!isnan(v)) { vsum += v; }
    }
    nn_cnt += __syncthreads_count(is_valid);
  }
  if (!t) {
    s->ck.non_nulls  = nn_cnt;
    s->ck.null_count = s->group.num_rows - nn_cnt;
  }
  vmin = warp_reduce(storage.float_stats[t / 32]).Reduce(vmin, hipcub::Min());
  vmin = shuffle(vmin);
  vmax = warp_reduce(storage.float_stats[t / 32]).Reduce(vmax, hipcub::Max());
  vmax = shuffle(vmax);
  vsum = warp_reduce(storage.float_stats[t / 32]).Reduce(vsum, IgnoreNaNSum());
  if (!(t & 0x1f)) {
    s->warp_min[t >> 5].fp_val = vmin;
    s->warp_max[t >> 5].fp_val = vmax;
    s->warp_sum[t >> 5].fp_val = vsum;
  }
  has_minmax = __syncthreads_or(vmin <= vmax);
  if (t < 32 * 1) {
    vmin = warp_reduce(storage.float_stats[t / 32]).Reduce(s->warp_min[t].fp_val, hipcub::Min());
    if (!(t & 0x1f)) {
      s->ck.min_value.fp_val = (vmin != 0.0) ? vmin : HIP_NEG_ZERO;
      s->ck.has_minmax       = (has_minmax);
    }
  } else if (t < 32 * 2) {
    vmax =
      warp_reduce(storage.float_stats[t / 32]).Reduce(s->warp_max[t & 0x1f].fp_val, hipcub::Max());
    if (!(t & 0x1f)) { s->ck.max_value.fp_val = (vmax != 0.0) ? vmax : HIP_ZERO; }
  } else if (t < 32 * 3) {
    vsum =
      warp_reduce(storage.float_stats[t / 32]).Reduce(s->warp_sum[t & 0x1f].fp_val, IgnoreNaNSum());
    if (!(t & 0x1f)) {
      s->ck.sum.fp_val = vsum;
      s->ck.has_sum    = (has_minmax);  // Implies sum is valid as well
    }
  }
}

// FIXME: Use native libcudf string type
struct nvstrdesc_s {
  const char *ptr;
  size_t count;
};

/**
 * @brief Gather statistics for string columns
 *
 * @param s shared block state
 * @param t thread id
 * @param storage temporary storage for warp reduction
 */
template <typename Storage>
void __device__ gatherStringColumnStats(stats_state_s *s, uint32_t t, Storage &storage)
{
  using warp_reduce = hipcub::WarpReduce<uint32_t>;
  uint32_t len_sum  = 0;
  const char *smin  = nullptr;
  const char *smax  = nullptr;
  uint32_t lmin     = 0;
  uint32_t lmax     = 0;
  uint32_t nn_cnt   = 0;
  bool has_minmax;
  string_stats minval, maxval;

  for (uint32_t i = 0; i < s->group.num_rows; i += 1024) {
    uint32_t r                = i + t;
    uint32_t row              = r + s->group.start_row;
    const uint32_t *valid_map = s->col.valid_map_base;
    uint32_t is_valid         = (r < s->group.num_rows && row < s->col.num_values)
                          ? (valid_map) ? (valid_map[(row + s->col.column_offset) >> 5] >>
                                           ((row + s->col.column_offset) & 0x1f)) &
                                            1
                                        : 1
                          : 0;
    if (is_valid) {
      const nvstrdesc_s *str_col = static_cast<const nvstrdesc_s *>(s->col.column_data_base);
      uint32_t len               = (uint32_t)str_col[row].count;
      const char *ptr            = str_col[row].ptr;
      len_sum += len;
      if (!smin || nvstr_is_lesser(ptr, len, smin, lmin)) {
        lmin = len;
        smin = ptr;
      }
      if (!smax || nvstr_is_greater(ptr, len, smax, lmax)) {
        lmax = len;
        smax = ptr;
      }
    }
    nn_cnt += __syncthreads_count(is_valid);
  }
  if (!t) {
    s->ck.non_nulls  = nn_cnt;
    s->ck.null_count = s->group.num_rows - nn_cnt;
  }
  minval  = WarpReduceMinString(smin, lmin);
  maxval  = WarpReduceMaxString(smax, lmax);
  len_sum = warp_reduce(storage.string_stats[t / 32]).Sum(len_sum);
  __syncwarp();
  if (!(t & 0x1f)) {
    s->warp_min[t >> 5].str_val.ptr    = minval.ptr;
    s->warp_min[t >> 5].str_val.length = minval.length;
    s->warp_max[t >> 5].str_val.ptr    = maxval.ptr;
    s->warp_max[t >> 5].str_val.length = maxval.length;
    s->warp_sum[t >> 5].str_val.length = len_sum;
  }
  has_minmax = __syncthreads_or(smin != nullptr);
  if (t < 32 * 1) {
    minval = WarpReduceMinString(s->warp_min[t].str_val.ptr, s->warp_min[t].str_val.length);
    if (!(t & 0x1f)) {
      s->ck.min_value.str_val.ptr    = minval.ptr;
      s->ck.min_value.str_val.length = minval.length;
      s->ck.has_minmax               = has_minmax;
    }
  } else if (t < 32 * 2) {
    maxval =
      WarpReduceMaxString(s->warp_max[t & 0x1f].str_val.ptr, s->warp_max[t & 0x1f].str_val.length);
    if (!(t & 0x1f)) {
      s->ck.max_value.str_val.ptr    = maxval.ptr;
      s->ck.max_value.str_val.length = maxval.length;
    }
  } else if (t < 32 * 3) {
    len_sum = warp_reduce(storage.string_stats[t / 32]).Sum(s->warp_sum[t & 0x1f].str_val.length);
    if (!(t & 0x1f)) {
      s->ck.sum.i_val = len_sum;
      s->ck.has_sum   = has_minmax;
    }
  }
}

/**
 * @brief Gather column chunk statistics (min/max values, sum and null count)
 * for a group of rows.
 *
 * blockDim {1024,1,1}
 *
 * @param chunks Destination statistics results
 * @param groups Statistics source information
 */
template <int block_size>
__global__ void __launch_bounds__(block_size, 1)
  gpuGatherColumnStatistics(statistics_chunk *chunks, const statistics_group *groups)
{
  __shared__ __align__(8) stats_state_s state_g;
  __shared__ union {
    typename hipcub::WarpReduce<int64_t>::TempStorage integer_stats[block_size / 32];
    typename hipcub::WarpReduce<double>::TempStorage float_stats[block_size / 32];
    typename hipcub::WarpReduce<uint32_t>::TempStorage string_stats[block_size / 32];
  } temp_storage;

  stats_state_s *const s = &state_g;
  uint32_t t             = threadIdx.x;
  statistics_dtype dtype;

  if (t < sizeof(statistics_group) / sizeof(uint32_t)) {
    reinterpret_cast<uint32_t *>(&s->group)[t] =
      reinterpret_cast<const uint32_t *>(&groups[blockIdx.x])[t];
  }
  if (t < sizeof(statistics_chunk) / sizeof(uint32_t)) {
    reinterpret_cast<uint32_t *>(&s->ck)[t] = 0;
  }
  __syncthreads();
  if (t < sizeof(stats_column_desc) / sizeof(uint32_t)) {
    reinterpret_cast<uint32_t *>(&s->col)[t] = reinterpret_cast<const uint32_t *>(s->group.col)[t];
  }
  __syncthreads();
  dtype = s->col.stats_dtype;
  if (dtype >= dtype_bool && dtype <= dtype_decimal64) {
    gatherIntColumnStats(s, dtype, t, temp_storage);
  } else if (dtype >= dtype_float32 && dtype <= dtype_float64) {
    gatherFloatColumnStats(s, dtype, t, temp_storage);
  } else if (dtype == dtype_string) {
    gatherStringColumnStats(s, t, temp_storage);
  }
  __syncthreads();
  if (t < sizeof(statistics_chunk) / sizeof(uint32_t)) {
    reinterpret_cast<uint32_t *>(&chunks[blockIdx.x])[t] = reinterpret_cast<uint32_t *>(&s->ck)[t];
  }
}

/**
 * @brief Merge statistics for integer-like columns
 *
 * @param s shared block state
 * @param dtype data type
 * @param ck_in pointer to first statistic chunk
 * @param num_chunks number of statistic chunks to merge
 * @param t thread id
 * @param storage temporary storage for warp reduction
 */
template <typename Storage>
void __device__ mergeIntColumnStats(merge_state_s *s,
                                    statistics_dtype dtype,
                                    const statistics_chunk *ck_in,
                                    uint32_t num_chunks,
                                    uint32_t t,
                                    Storage &storage)
{
  int64_t vmin        = INT64_MAX;
  int64_t vmax        = INT64_MIN;
  int64_t vsum        = 0;
  uint32_t non_nulls  = 0;
  uint32_t null_count = 0;
  bool has_minmax;
  for (uint32_t i = t; i < num_chunks; i += 1024) {
    const statistics_chunk *ck = &ck_in[i];
    if (ck->has_minmax) {
      vmin = min(vmin, ck->min_value.i_val);
      vmax = max(vmax, ck->max_value.i_val);
    }
    if (ck->has_sum) { vsum += ck->sum.i_val; }
    non_nulls += ck->non_nulls;
    null_count += ck->null_count;
  }
  non_nulls = hipcub::WarpReduce<uint32_t>(storage.u32[t / 32]).Sum(non_nulls);
  __syncwarp();
  vmin = hipcub::WarpReduce<int64_t>(storage.i64[t / 32]).Reduce(vmin, hipcub::Min());
  __syncwarp();
  vmin = shuffle(vmin);

  null_count = hipcub::WarpReduce<uint32_t>(storage.u32[t / 32]).Sum(null_count);
  __syncwarp();
  vmax = hipcub::WarpReduce<int64_t>(storage.i64[t / 32]).Reduce(vmax, hipcub::Max());
  __syncwarp();
  vmax = shuffle(vmax);

  vsum = hipcub::WarpReduce<int64_t>(storage.i64[t / 32]).Sum(vsum);

  if (!(t & 0x1f)) {
    s->warp_non_nulls[t >> 5] = non_nulls;
    s->warp_nulls[t >> 5]     = null_count;
    s->warp_min[t >> 5].i_val = vmin;
    s->warp_max[t >> 5].i_val = vmax;
    s->warp_sum[t >> 5].i_val = vsum;
  }
  has_minmax = __syncthreads_or(vmin <= vmax);
  if (t < 32 * 1) {
    vmin = hipcub::WarpReduce<int64_t>(storage.i64[t / 32]).Reduce(s->warp_min[t].i_val, hipcub::Min());
    if (!(t & 0x1f)) {
      s->ck.min_value.i_val = vmin;
      s->ck.has_minmax      = (has_minmax);
    }
  } else if (t < 32 * 2) {
    vmax =
      hipcub::WarpReduce<int64_t>(storage.i64[t / 32]).Reduce(s->warp_max[t & 0x1f].i_val, hipcub::Max());
    if (!(t & 0x1f)) { s->ck.max_value.i_val = vmax; }
  } else if (t < 32 * 3) {
    vsum = hipcub::WarpReduce<int64_t>(storage.i64[t / 32]).Sum(s->warp_sum[t & 0x1f].i_val);
    if (!(t & 0x1f)) {
      s->ck.sum.i_val = vsum;
      // TODO: For now, don't set the sum flag with 64-bit values so we don't have to check for
      // 64-bit sum overflow
      s->ck.has_sum = (dtype <= dtype_int32 && has_minmax);
    }
  } else if (t < 32 * 4) {
    non_nulls = hipcub::WarpReduce<uint32_t>(storage.u32[t / 32]).Sum(s->warp_non_nulls[t & 0x1f]);
    if (!(t & 0x1f)) { s->ck.non_nulls = non_nulls; }
  } else if (t < 32 * 5) {
    null_count = hipcub::WarpReduce<uint32_t>(storage.u32[t / 32]).Sum(s->warp_nulls[t & 0x1f]);
    if (!(t & 0x1f)) { s->ck.null_count = null_count; }
  }
}

/**
 * @brief Merge statistics for floating-point columns
 *
 * @param s shared block state
 * @param dtype data type
 * @param ck_in pointer to first statistic chunk
 * @param num_chunks number of statistic chunks to merge
 * @param t thread id
 * @param storage temporary storage for warp reduction
 */
template <typename Storage>
void __device__ mergeFloatColumnStats(merge_state_s *s,
                                      const statistics_chunk *ck_in,
                                      uint32_t num_chunks,
                                      uint32_t t,
                                      Storage &storage)
{
  double vmin         = HIP_INF;
  double vmax         = -HIP_INF;
  double vsum         = 0;
  uint32_t non_nulls  = 0;
  uint32_t null_count = 0;
  bool has_minmax;
  for (uint32_t i = t; i < num_chunks; i += 1024) {
    const statistics_chunk *ck = &ck_in[i];
    if (ck->has_minmax) {
      double v0 = ck->min_value.fp_val;
      double v1 = ck->max_value.fp_val;
      if (v0 < vmin) { vmin = v0; }
      if (v1 > vmax) { vmax = v1; }
    }
    if (ck->has_sum) { vsum += ck->sum.fp_val; }
    non_nulls += ck->non_nulls;
    null_count += ck->null_count;
  }

  non_nulls = hipcub::WarpReduce<uint32_t>(storage.u32[t / 32]).Sum(non_nulls);
  __syncwarp();
  vmin = hipcub::WarpReduce<double>(storage.f64[t / 32]).Reduce(vmin, hipcub::Min());
  __syncwarp();
  vmin = shuffle(vmin);

  null_count = hipcub::WarpReduce<uint32_t>(storage.u32[t / 32]).Sum(null_count);
  __syncwarp();
  vmax = hipcub::WarpReduce<double>(storage.f64[t / 32]).Reduce(vmax, hipcub::Max());
  __syncwarp();
  vmax = shuffle(vmax);

  vsum = hipcub::WarpReduce<double>(storage.f64[t / 32]).Reduce(vsum, IgnoreNaNSum());

  if (!(t & 0x1f)) {
    s->warp_non_nulls[t >> 5]  = non_nulls;
    s->warp_nulls[t >> 5]      = null_count;
    s->warp_min[t >> 5].fp_val = vmin;
    s->warp_max[t >> 5].fp_val = vmax;
    s->warp_sum[t >> 5].fp_val = vsum;
  }
  has_minmax = __syncthreads_or(vmin <= vmax);
  if (t < 32 * 1) {
    vmin = hipcub::WarpReduce<double>(storage.f64[t / 32]).Reduce(s->warp_min[t].fp_val, hipcub::Min());
    if (!(t & 0x1f)) {
      s->ck.min_value.fp_val = (vmin != 0.0) ? vmin : HIP_NEG_ZERO;
      s->ck.has_minmax       = (has_minmax);
    }
  } else if (t < 32 * 2) {
    vmax =
      hipcub::WarpReduce<double>(storage.f64[t / 32]).Reduce(s->warp_max[t & 0x1f].fp_val, hipcub::Max());
    if (!(t & 0x1f)) { s->ck.max_value.fp_val = (vmax != 0.0) ? vmax : HIP_ZERO; }
  } else if (t < 32 * 3) {
    vsum = hipcub::WarpReduce<double>(storage.f64[t / 32])
             .Reduce(s->warp_sum[t & 0x1f].fp_val, IgnoreNaNSum());
    if (!(t & 0x1f)) {
      s->ck.sum.fp_val = vsum;
      s->ck.has_sum    = (has_minmax);  // Implies sum is valid as well
    }
  } else if (t < 32 * 4) {
    non_nulls = hipcub::WarpReduce<uint32_t>(storage.u32[t / 32]).Sum(s->warp_non_nulls[t & 0x1f]);
    if (!(t & 0x1f)) { s->ck.non_nulls = non_nulls; }
  } else if (t < 32 * 5) {
    null_count = hipcub::WarpReduce<uint32_t>(storage.u32[t / 32]).Sum(s->warp_nulls[t & 0x1f]);
    if (!(t & 0x1f)) { s->ck.null_count = null_count; }
  }
}

/**
 * @brief Merge statistics for string columns
 *
 * @param s shared block state
 * @param ck_in pointer to first statistic chunk
 * @param num_chunks number of statistic chunks to merge
 * @param t thread id
 * @param storage temporary storage for warp reduction
 */
template <typename Storage>
void __device__ mergeStringColumnStats(merge_state_s *s,
                                       const statistics_chunk *ck_in,
                                       uint32_t num_chunks,
                                       uint32_t t,
                                       Storage &storage)
{
  uint32_t len_sum    = 0;
  const char *smin    = nullptr;
  const char *smax    = nullptr;
  uint32_t lmin       = 0;
  uint32_t lmax       = 0;
  uint32_t non_nulls  = 0;
  uint32_t null_count = 0;
  bool has_minmax;
  string_stats minval, maxval;

  for (uint32_t i = t; i < num_chunks; i += 1024) {
    const statistics_chunk *ck = &ck_in[i];
    if (ck->has_minmax) {
      uint32_t len0    = ck->min_value.str_val.length;
      const char *ptr0 = ck->min_value.str_val.ptr;
      uint32_t len1    = ck->max_value.str_val.length;
      const char *ptr1 = ck->max_value.str_val.ptr;
      if (!smin || (ptr0 && nvstr_is_lesser(ptr0, len0, smin, lmin))) {
        lmin = len0;
        smin = ptr0;
      }
      if (!smax || (ptr1 && nvstr_is_greater(ptr1, len1, smax, lmax))) {
        lmax = len1;
        smax = ptr1;
      }
    }
    if (ck->has_sum) { len_sum += (uint32_t)ck->sum.i_val; }
    non_nulls += ck->non_nulls;
    null_count += ck->null_count;
  }
  non_nulls = hipcub::WarpReduce<uint32_t>(storage.u32[t / 32]).Sum(non_nulls);
  __syncwarp();
  null_count = hipcub::WarpReduce<uint32_t>(storage.u32[t / 32]).Sum(null_count);
  __syncwarp();
  minval  = WarpReduceMinString(smin, lmin);
  maxval  = WarpReduceMaxString(smax, lmax);
  len_sum = hipcub::WarpReduce<uint32_t>(storage.u32[t / 32]).Sum(len_sum);
  if (!(t & 0x1f)) {
    s->warp_non_nulls[t >> 5]          = non_nulls;
    s->warp_nulls[t >> 5]              = null_count;
    s->warp_min[t >> 5].str_val.ptr    = minval.ptr;
    s->warp_min[t >> 5].str_val.length = minval.length;
    s->warp_max[t >> 5].str_val.ptr    = maxval.ptr;
    s->warp_max[t >> 5].str_val.length = maxval.length;
    s->warp_sum[t >> 5].str_val.length = len_sum;
  }
  has_minmax = __syncthreads_or(smin != nullptr);
  if (t < 32 * 1) {
    minval = WarpReduceMinString(s->warp_min[t].str_val.ptr, s->warp_min[t].str_val.length);
    if (!(t & 0x1f)) {
      s->ck.min_value.str_val.ptr    = minval.ptr;
      s->ck.min_value.str_val.length = minval.length;
      s->ck.has_minmax               = has_minmax;
    }
  } else if (t < 32 * 2) {
    maxval =
      WarpReduceMaxString(s->warp_max[t & 0x1f].str_val.ptr, s->warp_max[t & 0x1f].str_val.length);
    if (!(t & 0x1f)) {
      s->ck.max_value.str_val.ptr    = maxval.ptr;
      s->ck.max_value.str_val.length = maxval.length;
    }
  } else if (t < 32 * 3) {
    len_sum =
      hipcub::WarpReduce<uint32_t>(storage.u32[t / 32]).Sum(s->warp_sum[t & 0x1f].str_val.length);
    if (!(t & 0x1f)) {
      s->ck.sum.i_val = len_sum;
      s->ck.has_sum   = has_minmax;
    }
  } else if (t < 32 * 4) {
    non_nulls = hipcub::WarpReduce<uint32_t>(storage.u32[t / 32]).Sum(s->warp_non_nulls[t & 0x1f]);
    if (!(t & 0x1f)) { s->ck.non_nulls = non_nulls; }
  } else if (t < 32 * 5) {
    null_count = hipcub::WarpReduce<uint32_t>(storage.u32[t / 32]).Sum(s->warp_nulls[t & 0x1f]);
    if (!(t & 0x1f)) { s->ck.null_count = null_count; }
  }
}

/**
 * @brief Combine multiple statistics chunk together to form new statistics chunks
 *
 * blockDim {1024,1,1}
 *
 * @param chunks_out Destination statistic chunks
 * @param chunks_in Source statistic chunks
 * @param groups Statistic chunk grouping information
 */
template <int block_size>
__global__ void __launch_bounds__(block_size, 1)
  gpuMergeColumnStatistics(statistics_chunk *chunks_out,
                           const statistics_chunk *chunks_in,
                           const statistics_merge_group *groups)
{
  __shared__ __align__(8) merge_state_s state_g;
  __shared__ struct {
    typename hipcub::WarpReduce<uint32_t>::TempStorage u32[block_size / 32];
    typename hipcub::WarpReduce<int64_t>::TempStorage i64[block_size / 32];
    typename hipcub::WarpReduce<double>::TempStorage f64[block_size / 32];
  } storage;

  merge_state_s *const s = &state_g;
  uint32_t t             = threadIdx.x;
  statistics_dtype dtype;

  if (t < sizeof(statistics_merge_group) / sizeof(uint32_t)) {
    reinterpret_cast<uint32_t *>(&s->group)[t] =
      reinterpret_cast<const uint32_t *>(&groups[blockIdx.x])[t];
  }
  __syncthreads();
  if (t < sizeof(stats_column_desc) / sizeof(uint32_t)) {
    reinterpret_cast<uint32_t *>(&s->col)[t] = reinterpret_cast<const uint32_t *>(s->group.col)[t];
  }
  __syncthreads();
  dtype = s->col.stats_dtype;

  if (dtype >= dtype_bool && dtype <= dtype_decimal64) {
    mergeIntColumnStats(
      s, dtype, chunks_in + s->group.start_chunk, s->group.num_chunks, t, storage);
  } else if (dtype >= dtype_float32 && dtype <= dtype_float64) {
    mergeFloatColumnStats(s, chunks_in + s->group.start_chunk, s->group.num_chunks, t, storage);
  } else if (dtype == dtype_string) {
    mergeStringColumnStats(s, chunks_in + s->group.start_chunk, s->group.num_chunks, t, storage);
  }

  __syncthreads();
  if (t < sizeof(statistics_chunk) / sizeof(uint32_t)) {
    reinterpret_cast<uint32_t *>(&chunks_out[blockIdx.x])[t] =
      reinterpret_cast<uint32_t *>(&s->ck)[t];
  }
}

/**
 * @brief Launches kernel to gather column statistics
 *
 * @param[out] chunks Statistics results [num_chunks]
 * @param[in] groups Statistics row groups [num_chunks]
 * @param[in] num_chunks Number of chunks & rowgroups
 * @param[in] stream CUDA stream to use, default 0
 */
void GatherColumnStatistics(statistics_chunk *chunks,
                            const statistics_group *groups,
                            uint32_t num_chunks,
                            rmm::cuda_stream_view stream)
{
  gpuGatherColumnStatistics<1024><<<num_chunks, 1024, 0, stream.value()>>>(chunks, groups);
}

/**
 * @brief Launches kernel to merge column statistics
 *
 * @param[out] chunks_out Statistics results [num_chunks]
 * @param[out] chunks_in Input statistics
 * @param[in] groups Statistics groups [num_chunks]
 * @param[in] num_chunks Number of chunks & groups
 * @param[in] stream CUDA stream to use, default 0
 */
void MergeColumnStatistics(statistics_chunk *chunks_out,
                           const statistics_chunk *chunks_in,
                           const statistics_merge_group *groups,
                           uint32_t num_chunks,
                           rmm::cuda_stream_view stream)
{
  gpuMergeColumnStatistics<1024>
    <<<num_chunks, 1024, 0, stream.value()>>>(chunks_out, chunks_in, groups);
}

}  // namespace io
}  // namespace cudf
