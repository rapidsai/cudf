#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2018-2020, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

/** @file gpuinflate.cu

  Derived from zlib's contrib/puff.c, original copyright notice below

*/

/*
Copyright (C) 2002-2013 Mark Adler, all rights reserved
version 2.3, 21 Jan 2013

This software is provided 'as-is', without any express or implied
warranty.  In no event will the author be held liable for any damages
arising from the use of this software.

Permission is granted to anyone to use this software for any purpose,
including commercial applications, and to alter it and redistribute it
freely, subject to the following restrictions:

1. The origin of this software must not be misrepresented; you must not
claim that you wrote the original software. If you use this software
in a product, an acknowledgment in the product documentation would be
appreciated but is not required.
2. Altered source versions must be plainly marked as such, and must not be
misrepresented as being the original software.
3. This notice may not be removed or altered from any source distribution.

Mark Adler    madler@alumni.caltech.edu
*/

#include "gpuinflate.h"
#include "io_uncomp.h"

#include <io/utilities/block_utils.cuh>

#include <rmm/cuda_stream_view.hpp>

namespace cudf {
namespace io {

constexpr int max_bits    = 15;   // maximum bits in a code
constexpr int max_l_codes = 286;  // maximum number of literal/length codes
constexpr int max_d_codes = 30;   // maximum number of distance codes
constexpr int fix_l_codes = 288;  // number of fixed literal/length codes

constexpr int log2_len_lut  = 10;
constexpr int log2_dist_lut = 8;

/**
 * @brief Intermediate arrays for building huffman tables
 */
struct scratch_arr {
  int16_t lengths[max_l_codes + max_d_codes];  ///< descriptor code lengths
  int16_t offs[max_bits + 1];                  ///< offset in symbol table for each length (scratch)
};

/**
 * @brief Huffman LUTs for length and distance codes
 */
struct lut_arr {
  int32_t lenlut[1 << log2_len_lut];    ///< LUT for length decoding
  int32_t distlut[1 << log2_dist_lut];  ///< LUT for fast distance decoding
};

/// 4 batches of 32 symbols
constexpr int log2_batch_count = 2;  // 1..5
constexpr int log2_batch_size  = 5;
constexpr int batch_count      = (1 << log2_batch_count);
constexpr int batch_size       = (1 << log2_batch_size);

/**
 * @brief Inter-warp communication queue
 */
struct xwarp_s {
  int32_t batch_len[batch_count];  //< Length of each batch - <0:end, 0:not ready, >0:symbol count
  union {
    uint32_t symqueue[batch_count * batch_size];
    uint8_t symqueue8[batch_count * batch_size * 4];
  } u;
};

#define ENABLE_PREFETCH 1

#if ENABLE_PREFETCH
constexpr int log2_prefetch_size = 9;  // Must be at least LOG2_BATCH_SIZE+3
constexpr int prefetch_size      = (1 << log2_prefetch_size);

/// @brief Prefetcher state
struct prefetch_queue_s {
  const uint8_t *cur_p;  ///< Prefetch location
  int run;               ///< prefetcher will exit when run=0
  uint8_t pref_data[prefetch_size];
};

template <typename T>
inline __device__ volatile uint32_t *prefetch_addr32(volatile prefetch_queue_s &q, T *ptr)
{
  return reinterpret_cast<volatile uint32_t *>(&q.pref_data[(prefetch_size - 4) & (size_t)(ptr)]);
}

#endif  // ENABLE_PREFETCH

/**
 * @brief Inflate decompressor state
 */
struct inflate_state_s {
  // output state
  uint8_t *out;      ///< output buffer
  uint8_t *outbase;  ///< start of output buffer
  uint8_t *outend;   ///< end of output buffer
  // Input state
  uint8_t *cur;  ///< input buffer
  uint8_t *end;  ///< end of input buffer

  uint2 bitbuf;     ///< bit buffer (64-bit)
  uint32_t bitpos;  ///< position in bit buffer

  int32_t err;              ///< Error status
  int btype;                ///< current block type
  int blast;                ///< last block
  uint32_t stored_blk_len;  ///< length of stored (uncompressed) block

  uint16_t first_slow_len;  ///< first code not in fast LUT
  uint16_t index_slow_len;
  uint16_t first_slow_dist;
  uint16_t index_slow_dist;

  volatile xwarp_s x;
#if ENABLE_PREFETCH
  volatile prefetch_queue_s pref;
#endif

  int16_t lencnt[max_bits + 1];
  int16_t lensym[fix_l_codes];  // Assumes fix_l_codes >= max_l_codes
  int16_t distcnt[max_bits + 1];
  int16_t distsym[max_d_codes];

  union {
    scratch_arr scratch;
    lut_arr lut;
  } u;
};

inline __device__ unsigned int bfe(unsigned int source,
                                   unsigned int bit_start,
                                   unsigned int num_bits)
{
  unsigned int bits;
  asm("bfe.u32 %0, %1, %2, %3;" : "=r"(bits) : "r"(source), "r"(bit_start), "r"(num_bits));
  return bits;
};

inline __device__ uint32_t showbits(inflate_state_s *s, uint32_t n)
{
  uint32_t next32 = __funnelshift_rc(s->bitbuf.x, s->bitbuf.y, s->bitpos);
  return (next32 & ((1 << n) - 1));
}

inline __device__ uint32_t nextbits32(inflate_state_s *s)
{
  return __funnelshift_rc(s->bitbuf.x, s->bitbuf.y, s->bitpos);
}

inline __device__ void skipbits(inflate_state_s *s, uint32_t n)
{
  uint32_t bitpos = s->bitpos + n;
  if (bitpos >= 32) {
    uint8_t *cur = s->cur + 8;
    s->bitbuf.x  = s->bitbuf.y;
    s->bitbuf.y  = (cur < s->end) ? *reinterpret_cast<uint32_t *>(cur) : 0;
    s->cur       = cur - 4;
    bitpos &= 0x1f;
  }
  s->bitpos = bitpos;
}

// TODO: If we require 4-byte alignment of input bitstream & length (padded), reading bits would
// become quite a bit faster
__device__ uint32_t getbits(inflate_state_s *s, uint32_t n)
{
  uint32_t v = showbits(s, n);
  skipbits(s, n);
  return v;
}

/**
 * @brief Decode a code from the stream s using huffman table {symbols,counts}.
 * Return the symbol or a negative value if there is an error.
 * If all of the lengths are zero, i.e. an empty code, or if the code is
 * incomplete and an invalid code is received, then -10 is returned after
 * reading max_bits bits.
 *
 * Format notes:
 *
 * - The codes as stored in the compressed data are bit-reversed relative to
 *   a simple integer ordering of codes of the same lengths.  Hence below the
 *   bits are pulled from the compressed data one at a time and used to
 *   build the code value reversed from what is in the stream in order to
 *   permit simple integer comparisons for decoding.  A table-based decoding
 *   scheme (as used in zlib) does not need to do this reversal.
 *
 * - The first code for the shortest length is all zeros.  Subsequent codes of
 *   the same length are simply integer increments of the previous code.  When
 *   moving up a length, a zero bit is appended to the code.  For a complete
 *   code, the last code of the longest length will be all ones.
 *
 * - Incomplete codes are handled by this decoder, since they are permitted
 *   in the deflate format.  See the format notes for fixed() and dynamic().
 */
__device__ int decode(inflate_state_s *s, const int16_t *counts, const int16_t *symbols)
{
  unsigned int len;    // current number of bits in code
  unsigned int code;   // len bits being decoded
  unsigned int first;  // first code of length len
  unsigned int count;  // number of codes of length len
  uint32_t next32r = __brev(nextbits32(s));

  first = 0;
  for (len = 1; len <= max_bits; len++) {
    code  = (next32r >> (32 - len)) - first;
    count = counts[len];
    if (code < count)  // if length len, return symbol
    {
      skipbits(s, len);
      return symbols[code];
    }
    symbols += count;  // else update for next length
    first += count;
    first <<= 1;
  }
  return -10;  // ran out of codes
}

/**
 * @brief Given the list of code lengths length[0..n-1] representing a canonical
 * Huffman code for n symbols, construct the tables required to decode those
 * codes.  Those tables are the number of codes of each length, and the symbols
 * sorted by length, retaining their original order within each length.  The
 * return value is zero for a complete code set, negative for an over-
 * subscribed code set, and positive for an incomplete code set.  The tables
 * can be used if the return value is zero or positive, but they cannot be used
 * if the return value is negative.  If the return value is zero, it is not
 * possible for decode() using that table to return an error--any stream of
 * enough bits will resolve to a symbol.  If the return value is positive, then
 * it is possible for decode() using that table to return an error for received
 * codes past the end of the incomplete lengths.
 *
 * Not used by decode(), but used for error checking, count[0] is the number
 * of the n symbols not in the code.  So n - count[0] is the number of
 * codes.  This is useful for checking for incomplete codes that have more than
 * one symbol, which is an error in a dynamic block.
 *
 * Assumption: for all i in 0..n-1, 0 <= length[i] <= max_bits
 * This is assured by the construction of the length arrays in dynamic() and
 * fixed() and is not verified by construct().
 *
 * Format notes:
 *
 * - Permitted and expected examples of incomplete codes are one of the fixed
 *   codes and any code with a single symbol which in deflate is coded as one
 *   bit instead of zero bits.  See the format notes for fixed() and dynamic().
 *
 * - Within a given code length, the symbols are kept in ascending order for
 *   the code bits definition.
 */
__device__ int construct(
  inflate_state_s *s, int16_t *counts, int16_t *symbols, const int16_t *length, int n)
{
  int symbol;  // current symbol when stepping through length[]
  int len;     // current length when stepping through counts[]
  int left;    // number of possible codes left of current length
  int16_t *offs = s->u.scratch.offs;

  // count number of codes of each length
  for (len = 0; len <= max_bits; len++) counts[len] = 0;
  for (symbol = 0; symbol < n; symbol++)
    (counts[length[symbol]])++;  // assumes lengths are within bounds
  if (counts[0] == n)            // no codes!
    return 0;                    // complete, but decode() will fail

  // check for an over-subscribed or incomplete set of lengths
  left = 1;  // one possible code of zero length
  for (len = 1; len <= max_bits; len++) {
    left <<= 1;                 // one more bit, double codes left
    left -= counts[len];        // deduct count from possible codes
    if (left < 0) return left;  // over-subscribed--return negative
  }                             // left > 0 means incomplete

  // generate offsets into symbol table for each length for sorting
  offs[1] = 0;
  for (len = 1; len < max_bits; len++) offs[len + 1] = offs[len] + counts[len];

  // put symbols in table sorted by length, by symbol order within each length
  for (symbol = 0; symbol < n; symbol++)
    if (length[symbol] != 0) symbols[offs[length[symbol]]++] = symbol;

  // return zero for complete set, positive for incomplete set
  return left;
}

/// permutation of code length codes
static const __device__ __constant__ uint8_t g_code_order[19 + 1] = {
  16, 17, 18, 0, 8, 7, 9, 6, 10, 5, 11, 4, 12, 3, 13, 2, 14, 1, 15, 0xff};

/// Dynamic block (custom huffman tables)
__device__ int init_dynamic(inflate_state_s *s)
{
  int nlen, ndist, ncode; /* number of lengths in descriptor */
  int index;              /* index of lengths[] */
  int err;                /* construct() return value */
  int16_t *lengths = s->u.scratch.lengths;

  // get number of lengths in each table, check lengths
  nlen  = getbits(s, 5) + 257;
  ndist = getbits(s, 5) + 1;
  ncode = getbits(s, 4) + 4;
  if (nlen > max_l_codes || ndist > max_d_codes) {
    return -3;  // bad counts
  }
  // read code length code lengths (really), missing lengths are zero
  for (index = 0; index < ncode; index++) lengths[g_code_order[index]] = getbits(s, 3);
  for (; index < 19; index++) lengths[g_code_order[index]] = 0;

  // build huffman table for code lengths codes (use lencode temporarily)
  err = construct(s, s->lencnt, s->lensym, lengths, 19);
  if (err != 0)  // require complete code set here
    return -4;

  // read length/literal and distance code length tables
  index = 0;
  while (index < nlen + ndist) {
    int symbol = decode(s, s->lencnt, s->lensym);
    if (symbol < 0) return symbol;  // invalid symbol
    if (symbol < 16)                // length in 0..15
      lengths[index++] = symbol;
    else {                            // repeat instruction
      int len = 0;                    // last length to repeat, assume repeating zeros
      if (symbol == 16) {             // repeat last length 3..6 times
        if (index == 0) return -5;    // no last length!
        len    = lengths[index - 1];  // last length
        symbol = 3 + getbits(s, 2);
      } else if (symbol == 17)  // repeat zero 3..10 times
        symbol = 3 + getbits(s, 3);
      else  // == 18, repeat zero 11..138 times
        symbol = 11 + getbits(s, 7);
      if (index + symbol > nlen + ndist) return -6;  // too many lengths!
      while (symbol--)                               // repeat last or zero symbol times
        lengths[index++] = len;
    }
  }

  // check for end-of-block code -- there better be one!
  if (lengths[256] == 0) return -9;

  // build huffman table for literal/length codes
  err = construct(s, s->lencnt, s->lensym, lengths, nlen);
  if (err && (err < 0 || nlen != s->lencnt[0] + s->lencnt[1]))
    return -7;  // incomplete code ok only for single length 1 code

  // build huffman table for distance codes
  err = construct(s, s->distcnt, s->distsym, &lengths[nlen], ndist);
  if (err && (err < 0 || ndist != s->distcnt[0] + s->distcnt[1]))
    return -8;  // incomplete code ok only for single length 1 code

  return 0;
}

/**
 * @brief Initializes a fixed codes block.
 *
 * Format notes:
 *
 * - This block type can be useful for compressing small amounts of data for
 *   which the size of the code descriptions in a dynamic block exceeds the
 *   benefit of custom codes for that block.  For fixed codes, no bits are
 *   spent on code descriptions.  Instead the code lengths for literal/length
 *   codes and distance codes are fixed.  The specific lengths for each symbol
 *   can be seen in the "for" loops below.
 *
 * - The literal/length code is complete, but has two symbols that are invalid
 *   and should result in an error if received.  This cannot be implemented
 *   simply as an incomplete code since those two symbols are in the "middle"
 *   of the code.  They are eight bits long and the longest literal/length\
 *   code is nine bits.  Therefore the code must be constructed with those
 *   symbols, and the invalid symbols must be detected after decoding.
 *
 * - The fixed distance codes also have two invalid symbols that should result
 *   in an error if received.  Since all of the distance codes are the same
 *   length, this can be implemented as an incomplete code.  Then the invalid
 *   codes are detected while decoding.
 */
__device__ int init_fixed(inflate_state_s *s)
{
  int16_t *lengths = s->u.scratch.lengths;
  int symbol;

  // literal/length table
  for (symbol = 0; symbol < 144; symbol++) lengths[symbol] = 8;
  for (; symbol < 256; symbol++) lengths[symbol] = 9;
  for (; symbol < 280; symbol++) lengths[symbol] = 7;
  for (; symbol < fix_l_codes; symbol++) lengths[symbol] = 8;
  construct(s, s->lencnt, s->lensym, lengths, fix_l_codes);

  // distance table
  for (symbol = 0; symbol < max_d_codes; symbol++) lengths[symbol] = 5;

  // build huffman table for distance codes
  construct(s, s->distcnt, s->distsym, lengths, max_d_codes);

  return 0;
}

/**
 * @brief Decode literal/length and distance codes until an end-of-block code.
 *
 * Format notes:
 *
 * - Compressed data that is after the block type if fixed or after the code
 *   description if dynamic is a combination of literals and length/distance
 *   pairs terminated by and end-of-block code.  Literals are simply Huffman
 *   coded bytes.  A length/distance pair is a coded length followed by a
 *   coded distance to represent a string that occurs earlier in the
 *   uncompressed data that occurs again at the current location.
 *
 * - Literals, lengths, and the end-of-block code are combined into a single
 *   code of up to 286 symbols.  They are 256 literals (0..255), 29 length
 *   symbols (257..285), and the end-of-block symbol (256).
 *
 * - There are 256 possible lengths (3..258), and so 29 symbols are not enough
 *   to represent all of those.  Lengths 3..10 and 258 are in fact represented
 *   by just a length symbol.  Lengths 11..257 are represented as a symbol and
 *   some number of extra bits that are added as an integer to the base length
 *   of the length symbol.  The number of extra bits is determined by the base
 *   length symbol.  These are in the static arrays below, lens[] for the base
 *   lengths and lext[] for the corresponding number of extra bits.
 *
 * - The reason that 258 gets its own symbol is that the longest length is used
 *   often in highly redundant files.  Note that 258 can also be coded as the
 *   base value 227 plus the maximum extra value of 31.  While a good deflate
 *   should never do this, it is not an error, and should be decoded properly.
 *
 * - If a length is decoded, including its extra bits if any, then it is
 *   followed a distance code.  There are up to 30 distance symbols.  Again
 *   there are many more possible distances (1..32768), so extra bits are added
 *   to a base value represented by the symbol.  The distances 1..4 get their
 *   own symbol, but the rest require extra bits.  The base distances and
 *   corresponding number of extra bits are below in the static arrays dist[]
 *   and dext[].
 *
 * - Literal bytes are simply written to the output.  A length/distance pair is
 *   an instruction to copy previously uncompressed bytes to the output.  The
 *   copy is from distance bytes back in the output stream, copying for length
 *   bytes.
 *
 * - Distances pointing before the beginning of the output data are not
 *   permitted.
 *
 * - Overlapped copies, where the length is greater than the distance, are
 *   allowed and common.  For example, a distance of one and a length of 258
 *   simply copies the last byte 258 times.  A distance of four and a length of
 *   twelve copies the last four bytes three times.  A simple forward copy
 *   ignoring whether the length is greater than the distance or not implements
 *   this correctly.  You should not use memcpy() since its behavior is not
 *   defined for overlapped arrays.  You should not use memmove() or bcopy()
 *   since though their behavior -is- defined for overlapping arrays, it is
 *   defined to do the wrong thing in this case.
 */

/// permutation of code length codes
static const __device__ __constant__ uint16_t g_lens[29] = {  // Size base for length codes 257..285
  3,  4,  5,  6,  7,  8,  9,  10, 11,  13,  15,  17,  19,  23, 27,
  31, 35, 43, 51, 59, 67, 83, 99, 115, 131, 163, 195, 227, 258};
static const __device__ __constant__ uint16_t
  g_lext[29] = {  // Extra bits for length codes 257..285
    0, 0, 0, 0, 0, 0, 0, 0, 1, 1, 1, 1, 2, 2, 2, 2, 3, 3, 3, 3, 4, 4, 4, 4, 5, 5, 5, 5, 0};

static const __device__ __constant__ uint16_t
  g_dists[30] = {  // Offset base for distance codes 0..29
    1,   2,   3,   4,   5,   7,    9,    13,   17,   25,   33,   49,   65,    97,    129,
    193, 257, 385, 513, 769, 1025, 1537, 2049, 3073, 4097, 6145, 8193, 12289, 16385, 24577};
static const __device__ __constant__ uint16_t g_dext[30] = {  // Extra bits for distance codes 0..29
  0, 0, 0, 0, 1, 1, 2, 2, 3, 3, 4, 4, 5, 5, 6, 6, 7, 7, 8, 8, 9, 9, 10, 10, 11, 11, 12, 12, 13, 13};

/// @brief Thread 0 only: decode bitstreams and output symbols into the symbol queue
__device__ void decode_symbols(inflate_state_s *s)
{
  uint32_t bitpos = s->bitpos;
  uint2 bitbuf    = s->bitbuf;
  uint8_t *cur    = s->cur;
  uint8_t *end    = s->end;
  int32_t batch   = 0;
  int32_t sym, batch_len;

  do {
    volatile uint32_t *b = &s->x.u.symqueue[batch * batch_size];
    // Wait for the next batch entry to be empty
#if ENABLE_PREFETCH
    // Wait for prefetcher to fetch a worst-case of 48 bits per symbol
    while ((*(volatile int32_t *)&s->pref.cur_p - (int32_t)(size_t)cur < batch_size * 6) ||
           (s->x.batch_len[batch] != 0))
#else
    while (s->x.batch_len[batch] != 0)
#endif
    {
      nanosleep(100);
    }
    batch_len = 0;
#if ENABLE_PREFETCH
    if (cur + (bitpos >> 3) >= end) {
      s->err = 1;
      break;
    }
#endif
    // Inner loop decoding symbols
    do {
      uint32_t next32 = __funnelshift_rc(bitbuf.x, bitbuf.y, bitpos);  // nextbits32(s);
      uint32_t len;
      sym = s->u.lut.lenlut[next32 & ((1 << log2_len_lut) - 1)];
      if ((uint32_t)sym < (uint32_t)(0x100 << 5)) {
        // We can lookup a second symbol if this was a short literal
        len = sym & 0x1f;
        sym >>= 5;
        b[batch_len++] = sym;
        next32 >>= len;
        bitpos += len;
        sym = s->u.lut.lenlut[next32 & ((1 << log2_len_lut) - 1)];
      }
      if (sym > 0)  // short symbol
      {
        len = sym & 0x1f;
        sym = ((sym >> 5) & 0x3ff) + ((next32 >> (sym >> 24)) & ((sym >> 16) & 0x1f));
      } else {
        // Slow length path
        uint32_t next32r       = __brev(next32);
        const int16_t *symbols = &s->lensym[s->index_slow_len];
        unsigned int first     = s->first_slow_len;
        int lext;
#pragma unroll 1
        for (len = log2_len_lut + 1; len <= max_bits; len++) {
          unsigned int code  = (next32r >> (32 - len)) - first;
          unsigned int count = s->lencnt[len];
          if (code < count)  // if length len, return symbol
          {
            sym = symbols[code];
            break;
          }
          symbols += count;  // else update for next length
          first += count;
          first <<= 1;
        }
        if (len > max_bits) {
          s->err = -10;
          sym    = 256;
          len    = 0;
        }
        if (sym > 256) {
          sym -= 257;
          lext = g_lext[sym];
          sym  = 256 + g_lens[sym] + bfe(next32, len, lext);
          len += lext;
        }
      }
      if (sym > 256) {
        int dist, dext;
        // skipbits(s, len) inlined - no limit check
        bitpos += len;
        if (bitpos >= 32) {
          bitbuf.x = bitbuf.y;
#if ENABLE_PREFETCH
          bitbuf.y = *prefetch_addr32(s->pref, cur + 8);
          cur += 4;
#else
          cur += 8;
          bitbuf.y = (cur < end) ? *(const uint32_t *)cur : 0;
          cur -= 4;
#endif
          bitpos &= 0x1f;
        }
        // get distance
        next32 = __funnelshift_rc(bitbuf.x, bitbuf.y, bitpos);  // nextbits32(s);
        dist   = s->u.lut.distlut[next32 & ((1 << log2_dist_lut) - 1)];
        if (dist > 0) {
          len  = dist & 0x1f;
          dext = bfe(dist, 20, 5);
          dist = bfe(dist, 5, 15);
          sym |= (dist + bfe(next32, len, dext)) << 16;
          len += dext;
        } else {
          uint32_t next32r       = __brev(next32);
          const int16_t *symbols = &s->distsym[s->index_slow_dist];
          unsigned int first     = s->first_slow_dist;
#pragma unroll 1
          for (len = log2_dist_lut + 1; len <= max_bits; len++) {
            unsigned int code  = (next32r >> (32 - len)) - first;
            unsigned int count = s->distcnt[len];
            if (code < count)  // if length len, return symbol
            {
              dist = symbols[code];
              break;
            }
            symbols += count;  // else update for next length
            first += count;
            first <<= 1;
          }
          if (len > max_bits) {
            s->err = -10;
            sym    = 256;
            len    = 0;
          } else {
            dext = g_dext[dist];
            sym |= (g_dists[dist] + bfe(next32, len, dext)) << 16;
            len += dext;
          }
        }
      }
      // skipbits(s, len) inlined with added error check for reading past the end of the input
      // buffer
      bitpos += len;
      if (bitpos >= 32) {
        bitbuf.x = bitbuf.y;
#if ENABLE_PREFETCH
        bitbuf.y = *prefetch_addr32(s->pref, cur + 8);
        cur += 4;
#else
        cur += 8;
        if (cur < end) {
          bitbuf.y = *(const uint32_t *)cur;
          cur -= 4;
        } else {
          bitbuf.y = 0;
          cur -= 4;
          if (cur > end) {
            s->err = 1;
            sym    = 256;
          }
        }
#endif
        bitpos &= 0x1f;
      }
      if (sym == 256) break;
      b[batch_len++] = sym;
    } while (batch_len < batch_size - 1);
    s->x.batch_len[batch] = batch_len;
#if ENABLE_PREFETCH
    ((volatile inflate_state_s *)s)->cur = cur;
#endif
    if (batch_len != 0) batch = (batch + 1) & (batch_count - 1);
  } while (sym != 256);

  while (s->x.batch_len[batch] != 0) { nanosleep(150); }
  s->x.batch_len[batch] = -1;
  s->bitbuf             = bitbuf;
  s->bitpos             = bitpos;
#if !ENABLE_PREFETCH
  s->cur = cur;
#endif
}

/**
 * @brief Build lookup tables for faster decode
 * LUT format is symbols*16+length
 */
__device__ void init_length_lut(inflate_state_s *s, int t)
{
  int32_t *lut = s->u.lut.lenlut;

  for (uint32_t bits = t; bits < (1 << log2_len_lut); bits += blockDim.x) {
    const int16_t *cnt     = s->lencnt;
    const int16_t *symbols = s->lensym;
    int sym                = -10 << 5;
    unsigned int first     = 0;
    unsigned int rbits     = __brev(bits) >> (32 - log2_len_lut);
    for (unsigned int len = 1; len <= log2_len_lut; len++) {
      unsigned int code  = (rbits >> (log2_len_lut - len)) - first;
      unsigned int count = cnt[len];
      if (code < count) {
        sym = symbols[code];
        if (sym > 256) {
          int lext = g_lext[sym - 257];
          sym = (256 + g_lens[sym - 257]) | (((1 << lext) - 1) << (16 - 5)) | (len << (24 - 5));
          len += lext;
        }
        sym = (sym << 5) | len;
        break;
      }
      symbols += count;  // else update for next length
      first += count;
      first <<= 1;
    }
    lut[bits] = sym;
  }
  if (!t) {
    unsigned int first = 0;
    unsigned int index = 0;
    const int16_t *cnt = s->lencnt;
    for (unsigned int len = 1; len <= log2_len_lut; len++) {
      unsigned int count = cnt[len];
      index += count;
      first += count;
      first <<= 1;
    }
    s->first_slow_len = first;
    s->index_slow_len = index;
  }
}

/**
 * @brief Build lookup tables for faster decode of distance symbol
 * LUT format is symbols*16+length
 */
__device__ void init_distance_lut(inflate_state_s *s, int t)
{
  int32_t *lut = s->u.lut.distlut;

  for (uint32_t bits = t; bits < (1 << log2_dist_lut); bits += blockDim.x) {
    const int16_t *cnt     = s->distcnt;
    const int16_t *symbols = s->distsym;
    int sym                = 0;
    unsigned int first     = 0;
    unsigned int rbits     = __brev(bits) >> (32 - log2_dist_lut);
    for (unsigned int len = 1; len <= log2_dist_lut; len++) {
      unsigned int code  = (rbits >> (log2_dist_lut - len)) - first;
      unsigned int count = cnt[len];
      if (code < count) {
        int dist = symbols[code];
        int dext = g_dext[dist];
        sym      = g_dists[dist] | (dext << 15);
        sym      = (sym << 5) | len;
        break;
      }
      symbols += count;  // else update for next length
      first += count;
      first <<= 1;
    }
    lut[bits] = sym;
  }
  if (!t) {
    unsigned int first = 0;
    unsigned int index = 0;
    const int16_t *cnt = s->distcnt;
    for (unsigned int len = 1; len <= log2_dist_lut; len++) {
      unsigned int count = cnt[len];
      index += count;
      first += count;
      first <<= 1;
    }
    s->first_slow_dist = first;
    s->index_slow_dist = index;
  }
}

/// @brief WARP1: process symbols and output uncompressed stream
__device__ void process_symbols(inflate_state_s *s, int t)
{
  uint8_t *out           = s->out;
  const uint8_t *outend  = s->outend;
  const uint8_t *outbase = s->outbase;
  int batch              = 0;

  do {
    volatile uint32_t *b = &s->x.u.symqueue[batch * batch_size];
    int batch_len, pos;
    int32_t symt;
    uint32_t lit_mask;

    if (t == 0) {
      while ((batch_len = s->x.batch_len[batch]) == 0) { nanosleep(100); }
    } else {
      batch_len = 0;
    }
    batch_len = shuffle(batch_len);
    if (batch_len < 0) { break; }

    symt     = (t < batch_len) ? b[t] : 256;
    lit_mask = ballot(symt >= 256);
    pos      = min((__ffs(lit_mask) - 1) & 0xff, 32);
    if (t == 0) { s->x.batch_len[batch] = 0; }
    if (t < pos && out + t < outend) { out[t] = symt; }
    out += pos;
    batch_len -= pos;
    while (batch_len > 0) {
      int dist, len, symbol;

      // Process a non-literal symbol
      symbol = shuffle(symt, pos);
      len    = max((symbol & 0xffff) - 256, 0);  // max should be unnecessary, but just in case
      dist   = symbol >> 16;
      for (int i = t; i < len; i += 32) {
        const uint8_t *src = out + ((i >= dist) ? (i % dist) : i) - dist;
        uint8_t b          = (src < outbase) ? 0 : *src;
        if (out + i < outend) { out[i] = b; }
      }
      out += len;
      pos++;
      batch_len--;
      // Process subsequent literals, if any
      if (!((lit_mask >> pos) & 1)) {
        len    = min((__ffs(lit_mask >> pos) - 1) & 0xff, batch_len);
        symbol = shuffle(symt, (pos + t) & 0x1f);
        if (t < len && out + t < outend) { out[t] = symbol; }
        out += len;
        pos += len;
        batch_len -= len;
      }
    }
    batch = (batch + 1) & (batch_count - 1);
  } while (1);

  if (t == 0) { s->out = out; }
}

/**
 * @brief Initializes a stored block.
 *
 * Format notes:
 *
 * - After the two-bit stored block type (00), the stored block length and
 *   stored bytes are byte-aligned for fast copying.  Therefore any leftover
 *   bits in the byte that has the last bit of the type, as many as seven, are
 *   discarded.  The value of the discarded bits are not defined and should not
 *   be checked against any expectation.
 *
 * - The second inverted copy of the stored block length does not have to be
 *   checked, but it's probably a good idea to do so anyway.
 *
 * - A stored block can have zero length.  This is sometimes used to byte-align
 *   subsets of the compressed data for random access or partial recovery.
 */
__device__ int init_stored(inflate_state_s *s)
{
  uint32_t len, nlen;  // length of stored block

  // Byte align
  if (s->bitpos & 7) { skipbits(s, 8 - (s->bitpos & 7)); }
  if (s->cur + (s->bitpos >> 3) >= s->end) {
    return 2;  // Not enough input
  }
  // get length and check against its one's complement
  len  = getbits(s, 16);
  nlen = getbits(s, 16);
  if (len != (nlen ^ 0xffff)) {
    return -2;  // didn't match complement!
  }
  if (s->cur + (s->bitpos >> 3) + len > s->end) {
    return 2;  // Not enough input
  }
  s->stored_blk_len = len;

  // done with a valid stored block
  return 0;
}

/// Copy bytes from stored block to destination
__device__ void copy_stored(inflate_state_s *s, int t)
{
  int len         = s->stored_blk_len;
  uint8_t *cur    = s->cur + (s->bitpos >> 3);
  uint8_t *out    = s->out;
  uint8_t *outend = s->outend;
  uint8_t *cur4;
  int slow_bytes = min(len, (int)((16 - (size_t)out) & 0xf));
  int fast_bytes, bitpos;

  // Slow copy until output is 16B aligned
  if (slow_bytes) {
    for (int i = t; i < slow_bytes; i += blockDim.x) {
      if (out + i < outend) {
        out[i] = cur[i];  // Input range has already been validated in init_stored()
      }
    }
    cur += slow_bytes;
    out += slow_bytes;
    len -= slow_bytes;
  }
  fast_bytes = len;
  if (out < outend) { fast_bytes = (int)min((size_t)fast_bytes, (outend - out)); }
  fast_bytes &= ~0xf;
  bitpos = ((int)(3 & (size_t)cur)) << 3;
  cur4   = cur - (bitpos >> 3);
  if (out < outend) {
    // Fast copy 16 bytes at a time
    for (int i = t * 16; i < fast_bytes; i += blockDim.x * 16) {
      uint4 u;
      u.x = *reinterpret_cast<const uint32_t *>(cur4 + i + 0 * 4);
      u.y = *reinterpret_cast<const uint32_t *>(cur4 + i + 1 * 4);
      u.z = *reinterpret_cast<const uint32_t *>(cur4 + i + 2 * 4);
      u.w = *reinterpret_cast<const uint32_t *>(cur4 + i + 3 * 4);
      if (bitpos != 0) {
        uint32_t v = (bitpos != 0) ? *reinterpret_cast<const uint32_t *>(cur4 + i + 4 * 4) : 0;
        u.x        = __funnelshift_rc(u.x, u.y, bitpos);
        u.y        = __funnelshift_rc(u.y, u.z, bitpos);
        u.z        = __funnelshift_rc(u.z, u.w, bitpos);
        u.w        = __funnelshift_rc(u.w, v, bitpos);
      }
      *reinterpret_cast<uint4 *>(out + i) = u;
    }
  }
  cur += fast_bytes;
  out += fast_bytes;
  len -= fast_bytes;
  // Slow copy for remaining bytes
  for (int i = t; i < len; i += blockDim.x) {
    if (out + i < outend) {
      out[i] = cur[i];  // Input range has already been validated in init_stored()
    }
  }
  out += len;
  __syncthreads();
  if (t == 0) {
    // Reset bitstream to end of block
    uint8_t *p            = cur + len;
    uint32_t prefix_bytes = (uint32_t)(((size_t)p) & 3);
    p -= prefix_bytes;
    s->cur      = p;
    s->bitbuf.x = (p < s->end) ? *reinterpret_cast<uint32_t *>(p) : 0;
    p += 4;
    s->bitbuf.y = (p < s->end) ? *reinterpret_cast<uint32_t *>(p) : 0;
    s->bitpos   = prefix_bytes * 8;
    s->out      = out;
  }
}

#if ENABLE_PREFETCH
__device__ void init_prefetcher(inflate_state_s *s, int t)
{
  if (t == 0) {
    s->pref.cur_p = s->cur;
    s->pref.run   = 1;
  }
}

__device__ void prefetch_warp(volatile inflate_state_s *s, int t)
{
  const uint8_t *cur_p = s->pref.cur_p;
  const uint8_t *end   = s->end;
  while (shuffle((t == 0) ? s->pref.run : 0)) {
    int32_t cur_lo = (int32_t)(size_t)cur_p;
    int do_pref =
      shuffle((t == 0) ? (cur_lo - *(volatile int32_t *)&s->cur < prefetch_size - 32 * 4 - 4) : 0);
    if (do_pref) {
      const uint8_t *p             = cur_p + 4 * t;
      *prefetch_addr32(s->pref, p) = (p < end) ? *reinterpret_cast<const uint32_t *>(p) : 0;
      cur_p += 4 * 32;
      __threadfence_block();
      __syncwarp();
      if (!t) {
        s->pref.cur_p = cur_p;
        __threadfence_block();
      }
    } else if (t == 0) {
      nanosleep(150);
    }
  }
}
#endif  // ENABLE_PREFETCH

/**
 * @brief Parse GZIP header
 * See https://tools.ietf.org/html/rfc1952
 */
__device__ int parse_gzip_header(const uint8_t *src, size_t src_size)
{
  int hdr_len = -1;

  if (src_size >= 18) {
    uint32_t sig = (src[0] << 16) | (src[1] << 8) | src[2];
    if (sig == 0x1f8b08)  // 24-bit GZIP inflate signature {0x1f, 0x8b, 0x08}
    {
      uint8_t flags = src[3];
      hdr_len       = 10;
      if (flags & GZIPHeaderFlag::fextra)  // Extra fields present
      {
        int xlen = src[hdr_len] | (src[hdr_len + 1] << 8);
        hdr_len += xlen;
        if (hdr_len >= src_size) return -1;
      }
      if (flags & GZIPHeaderFlag::fname)  // Original file name present
      {
        // Skip zero-terminated string
        do {
          if (hdr_len >= src_size) return -1;
        } while (src[hdr_len++] != 0);
      }
      if (flags & GZIPHeaderFlag::fcomment)  // Comment present
      {
        // Skip zero-terminated string
        do {
          if (hdr_len >= src_size) return -1;
        } while (src[hdr_len++] != 0);
      }
      if (flags & GZIPHeaderFlag::fhcrc)  // Header CRC present
      {
        hdr_len += 2;
      }
      if (hdr_len + 8 >= src_size) hdr_len = -1;
    }
  }
  return hdr_len;
}

/**
 * @brief INFLATE decompression kernel
 *
 * blockDim {block_size,1,1}
 *
 * @tparam block_size Thread block dimension for this call
 * @param inputs Source and destination buffer information per block
 * @param outputs Decompression status buffer per block
 * @param parse_hdr If nonzero, indicates that the compressed bitstream includes a GZIP header
 */
template <int block_size>
__global__ void __launch_bounds__(block_size)
  inflate_kernel(gpu_inflate_input_s *inputs, gpu_inflate_status_s *outputs, int parse_hdr)
{
  __shared__ __align__(16) inflate_state_s state_g;

  int t                  = threadIdx.x;
  int z                  = blockIdx.x;
  inflate_state_s *state = &state_g;

  if (!t) {
    uint8_t *p      = const_cast<uint8_t *>(static_cast<uint8_t const *>(inputs[z].srcDevice));
    size_t src_size = inputs[z].srcSize;
    uint32_t prefix_bytes;
    // Parse header if needed
    state->err = 0;
    if (parse_hdr) {
      int hdr_len = parse_gzip_header(p, src_size);
      src_size    = (src_size >= 8) ? src_size - 8 : 0;  // ignore footer
      if (hdr_len >= 0) {
        p += hdr_len;
        src_size -= hdr_len;
      } else {
        state->err = hdr_len;
      }
    }
    // Initialize shared state
    state->out     = const_cast<uint8_t *>(static_cast<uint8_t const *>(inputs[z].dstDevice));
    state->outbase = state->out;
    state->outend  = state->out + inputs[z].dstSize;
    state->end     = p + src_size;
    prefix_bytes   = (uint32_t)(((size_t)p) & 3);
    p -= prefix_bytes;
    state->cur      = p;
    state->bitbuf.x = (p < state->end) ? *reinterpret_cast<uint32_t *>(p) : 0;
    p += 4;
    state->bitbuf.y = (p < state->end) ? *reinterpret_cast<uint32_t *>(p) : 0;
    state->bitpos   = prefix_bytes * 8;
  }
  __syncthreads();
  // Main loop decoding blocks
  while (!state->err) {
    if (!t) {
      // Thread0: read last flag, block type and custom huffman tables if any
      if (state->cur + (state->bitpos >> 3) >= state->end)
        state->err = 2;
      else {
        state->blast = getbits(state, 1);
        state->btype = getbits(state, 2);
        if (state->btype == 0)
          state->err = init_stored(state);
        else if (state->btype == 1)
          state->err = init_fixed(state);
        else if (state->btype == 2)
          state->err = init_dynamic(state);
        else
          state->err = -1;  // Invalid block
      }
    }
    __syncthreads();
    if (!state->err && (state->btype == 1 || state->btype == 2)) {
      // Initializes lookup tables (block wide)
      init_length_lut(state, t);
      init_distance_lut(state, t);
#if ENABLE_PREFETCH
      // Initialize prefetcher
      init_prefetcher(state, t);
#endif
      if (t < batch_count) { state->x.batch_len[t] = 0; }
      __syncthreads();
      // decode data until end-of-block code
      if (t < 1 * 32) {
        // WARP0: decode variable-length symbols
        if (!t) {
          // Thread0: decode symbols (single threaded)
          decode_symbols(state);
#if ENABLE_PREFETCH
          state->pref.run = 0;
#endif
        }
      } else if (t < 2 * 32) {
        // WARP1: perform LZ77 using length and distance codes from WARP0
        process_symbols(state, t & 0x1f);
      }
#if ENABLE_PREFETCH
      else if (t < 3 * 32) {
        // WARP2: Prefetcher: prefetch data for WARP0
        prefetch_warp(state, t & 0x1f);
      }
#endif
      // else WARP3: idle
    } else if (!state->err && state->btype == 0) {
      // Uncompressed block (block-wide memcpy)
      copy_stored(state, t);
    }
    if (state->blast) break;
    __syncthreads();
  }
  __syncthreads();
  // Output decompression status and length
  if (!t) {
    if (state->err == 0 && state->cur + ((state->bitpos + 7) >> 3) > state->end) {
      // Read past the end of the input buffer
      state->err = 2;
    } else if (state->err == 0 && state->out > state->outend) {
      // Output buffer too small
      state->err = 1;
    }
    outputs[z].bytes_written = state->out - state->outbase;
    outputs[z].status        = state->err;
    outputs[z].reserved      = (int)(state->end - state->cur);  // Here mainly for debug purposes
  }
}

/**
 * @brief Copy a group of buffers
 *
 * blockDim {1024,1,1}
 *
 * @param inputs Source and destination information per block
 */
__global__ void __launch_bounds__(1024) copy_uncompressed_kernel(gpu_inflate_input_s *inputs)
{
  __shared__ const uint8_t *volatile src_g;
  __shared__ uint8_t *volatile dst_g;
  __shared__ uint32_t volatile copy_len_g;

  uint32_t t = threadIdx.x;
  uint32_t z = blockIdx.x;
  const uint8_t *src;
  uint8_t *dst;
  uint32_t len, src_align_bytes, src_align_bits, dst_align_bytes;

  if (!t) {
    src        = static_cast<const uint8_t *>(inputs[z].srcDevice);
    dst        = static_cast<uint8_t *>(inputs[z].dstDevice);
    len        = min((uint32_t)inputs[z].srcSize, (uint32_t)inputs[z].dstSize);
    src_g      = src;
    dst_g      = dst;
    copy_len_g = len;
  }
  __syncthreads();
  src = src_g;
  dst = dst_g;
  len = copy_len_g;
  // Align output to 32-bit
  dst_align_bytes = 3 & -reinterpret_cast<intptr_t>(dst);
  if (dst_align_bytes != 0) {
    uint32_t align_len = min(dst_align_bytes, len);
    if (t < align_len) { dst[t] = src[t]; }
    src += align_len;
    dst += align_len;
    len -= align_len;
  }
  src_align_bytes = (uint32_t)(3 & reinterpret_cast<uintptr_t>(src));
  src_align_bits  = src_align_bytes << 3;
  while (len >= 32) {
    const uint32_t *src32 = reinterpret_cast<const uint32_t *>(src - src_align_bytes);
    uint32_t copy_cnt     = min(len >> 2, 1024);
    if (t < copy_cnt) {
      uint32_t v = src32[t];
      if (src_align_bits != 0) { v = __funnelshift_r(v, src32[t + 1], src_align_bits); }
      reinterpret_cast<uint32_t *>(dst)[t] = v;
    }
    src += copy_cnt * 4;
    dst += copy_cnt * 4;
    len -= copy_cnt * 4;
  }
  if (t < len) { dst[t] = src[t]; }
}

hipError_t __host__ gpuinflate(gpu_inflate_input_s *inputs,
                                gpu_inflate_status_s *outputs,
                                int count,
                                int parse_hdr,
                                rmm::cuda_stream_view stream)
{
  constexpr int block_size = 128;  // Threads per block
  if (count > 0) {
    inflate_kernel<block_size>
      <<<count, block_size, 0, stream.value()>>>(inputs, outputs, parse_hdr);
  }
  return hipSuccess;
}

hipError_t __host__ gpu_copy_uncompressed_blocks(gpu_inflate_input_s *inputs,
                                                  int count,
                                                  rmm::cuda_stream_view stream)
{
  if (count > 0) { copy_uncompressed_kernel<<<count, 1024, 0, stream.value()>>>(inputs); }
  return hipSuccess;
}

}  // namespace io
}  // namespace cudf
