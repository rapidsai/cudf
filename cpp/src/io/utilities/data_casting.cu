#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2022-2023, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <io/utilities/parsing_utils.cuh>
#include <io/utilities/string_parsing.hpp>

#include <cudf/column/column.hpp>
#include <cudf/column/column_device_view.cuh>
#include <cudf/column/column_factories.hpp>
#include <cudf/detail/null_mask.hpp>
#include <cudf/detail/nvtx/ranges.hpp>
#include <cudf/detail/utilities/cuda.cuh>
#include <cudf/detail/utilities/integer_utils.hpp>
#include <cudf/strings/detail/strings_children.cuh>
#include <cudf/strings/detail/utf8.hpp>
#include <cudf/types.hpp>

#include <rmm/cuda_stream_view.hpp>
#include <rmm/device_buffer.hpp>
#include <rmm/exec_policy.hpp>

#include <thrust/copy.h>
#include <thrust/functional.h>
#include <thrust/transform_reduce.h>

#include <hipcub/hipcub.hpp>

#include <memory>
#include <type_traits>

namespace cudf::io::json::detail {

constexpr auto SINGLE_THREAD_THRESHOLD = 128;
constexpr auto WARP_THRESHOLD          = 128 * 128;  // 16K

// Unicode code point escape sequence
static constexpr char UNICODE_SEQ = 0x7F;

// Invalid escape sequence
static constexpr char NON_ESCAPE_CHAR = 0x7E;

// Unicode code point escape sequence prefix comprises '\' and 'u' characters
static constexpr size_type UNICODE_ESC_PREFIX = 2;

// Unicode code point escape sequence comprises four hex characters
static constexpr size_type UNICODE_HEX_DIGIT_COUNT = 4;

// A unicode code point escape sequence is \uXXXX
static auto constexpr NUM_UNICODE_ESC_SEQ_CHARS = UNICODE_ESC_PREFIX + UNICODE_HEX_DIGIT_COUNT;

static constexpr auto UTF16_HIGH_SURROGATE_BEGIN = 0xD800;
static constexpr auto UTF16_HIGH_SURROGATE_END   = 0xDC00;
static constexpr auto UTF16_LOW_SURROGATE_BEGIN  = 0xDC00;
static constexpr auto UTF16_LOW_SURROGATE_END    = 0xE000;

/**
 * @brief Describing whether data casting of a certain item succeed, the item was parsed to null, or
 * whether type casting failed.
 */
enum class data_casting_result { PARSING_SUCCESS, PARSED_TO_NULL, PARSING_FAILURE };

/**
 * @brief Providing additional information about the type casting result.
 */
struct data_casting_result_info {
  // Number of bytes written to output
  size_type bytes;
  // Whether parsing succeeded, item was parsed to null, or failed
  data_casting_result result;
};

/**
 * @brief Returns the character to output for a given escaped character that's following a
 * backslash.
 *
 * @param escaped_char The character following the backslash.
 * @return The character to output for a given character that's following a backslash
 */
__device__ __forceinline__ char get_escape_char(char escaped_char)
{
  switch (escaped_char) {
    case '"': return '"';
    case '\\': return '\\';
    case '/': return '/';
    case 'b': return '\b';
    case 'f': return '\f';
    case 'n': return '\n';
    case 'r': return '\r';
    case 't': return '\t';
    case 'u': return UNICODE_SEQ;
    default: return NON_ESCAPE_CHAR;
  }
}

/**
 * @brief Parses the hex value from the four hex digits of a unicode code point escape sequence
 * \uXXXX.
 *
 * @param str Pointer to the first (most-significant) hex digit
 * @return The parsed hex value if successful, -1 otherwise.
 */
__device__ __forceinline__ int32_t parse_unicode_hex(char const* str)
{
  // Prepare result
  int32_t result = 0, base = 1;
  constexpr int32_t hex_radix = 16;

  // Iterate over hex digits right-to-left
  size_type index = UNICODE_HEX_DIGIT_COUNT;
  while (index-- > 0) {
    char const ch = str[index];
    if (ch >= '0' && ch <= '9') {
      result += static_cast<int32_t>((ch - '0') + 0) * base;
      base *= hex_radix;
    } else if (ch >= 'A' && ch <= 'F') {
      result += static_cast<int32_t>((ch - 'A') + 10) * base;
      base *= hex_radix;
    } else if (ch >= 'a' && ch <= 'f') {
      result += static_cast<int32_t>((ch - 'a') + 10) * base;
      base *= hex_radix;
    } else {
      return -1;
    }
  }
  return result;
}

/**
 * @brief Writes the UTF-8 byte sequence to \p out_it and returns the number of bytes written to
 * \p out_it
 */
constexpr size_type write_utf8_char(char_utf8 character, char*& out_it)
{
  auto const bytes = (out_it == nullptr) ? strings::detail::bytes_in_char_utf8(character)
                                         : strings::detail::from_char_utf8(character, out_it);
  if (out_it) out_it += bytes;
  return bytes;
}

/**
 * @brief Processes a string, replaces escape sequences and optionally strips off the quote
 * characters.
 *
 * @tparam in_iterator_t A bidirectional input iterator type whose value_type is convertible to
 * char
 * @param in_begin Iterator to the first item to process
 * @param in_end Iterator to one past the last item to process
 * @param d_buffer Output character buffer to the first item to write
 * @param options Settings for controlling string processing behavior
 * @return A struct of (num_bytes_written, parsing_success_result), where num_bytes_written is
 * the number of bytes written to d_buffer, parsing_success_result is enum value indicating whether
 * parsing succeeded, item was parsed to null, or failed.
 */
template <typename in_iterator_t>
__device__ __forceinline__ data_casting_result_info
process_string(in_iterator_t in_begin,
               in_iterator_t in_end,
               char* d_buffer,
               cudf::io::parse_options_view const& options)
{
  int32_t bytes           = 0;
  auto const num_in_chars = thrust::distance(in_begin, in_end);
  // String values are indicated by keeping the quote character
  bool const is_string_value =
    num_in_chars >= 2LL &&
    (options.quotechar == '\0' ||
     (*in_begin == options.quotechar) && (*thrust::prev(in_end) == options.quotechar));

  // Copy literal/numeric value
  if (not is_string_value) {
    bytes += (in_end - in_begin);
    if (d_buffer) d_buffer = thrust::copy(thrust::seq, in_begin, in_end, d_buffer);
    return {bytes, data_casting_result::PARSING_SUCCESS};
  }
  char constexpr backslash_char = '\\';

  // Escape-flag, set after encountering a backslash character
  bool is_prev_char_escape = false;

  // Exclude beginning and ending quote chars from string range
  if (!options.keepquotes) {
    ++in_begin;
    --in_end;
  }

  // Iterate over the input
  while (in_begin != in_end) {
    // Copy single character to output
    if (!is_prev_char_escape) {
      is_prev_char_escape = (*in_begin == backslash_char);
      if (!is_prev_char_escape) {
        if (d_buffer) *d_buffer++ = *in_begin;
        ++bytes;
      }
      ++in_begin;
      continue;
    }

    // Previous char indicated beginning of escape sequence
    // Reset escape flag for next loop iteration
    is_prev_char_escape = false;

    // Check the character that is supposed to be escaped
    auto escaped_char = get_escape_char(*in_begin);

    // We escaped an invalid escape character -> "fail"/null for this item
    if (escaped_char == NON_ESCAPE_CHAR) { return {bytes, data_casting_result::PARSING_FAILURE}; }

    // Regular, single-character escape
    if (escaped_char != UNICODE_SEQ) {
      if (d_buffer) *d_buffer++ = escaped_char;
      ++bytes;
      ++in_begin;
      continue;
    }

    // This is an escape sequence of a unicode code point: \uXXXX,
    // where each X in XXXX represents a hex digit
    // Skip over the 'u' char from \uXXXX to the first hex digit
    ++in_begin;

    // Make sure that there's at least 4 characters left from the
    // input, which are expected to be hex digits
    if (thrust::distance(in_begin, in_end) < UNICODE_HEX_DIGIT_COUNT) {
      return {bytes, data_casting_result::PARSING_FAILURE};
    }

    auto hex_val = parse_unicode_hex(in_begin);

    // Couldn't parse hex values from the four-character sequence -> "fail"/null for this item
    if (hex_val < 0) { return {bytes, data_casting_result::PARSING_FAILURE}; }

    // Skip over the four hex digits
    thrust::advance(in_begin, UNICODE_HEX_DIGIT_COUNT);

    // If this may be a UTF-16 encoded surrogate pair:
    // we expect another \uXXXX sequence
    int32_t hex_low_val = 0;
    if (hex_val >= UTF16_HIGH_SURROGATE_BEGIN && hex_val < UTF16_HIGH_SURROGATE_END &&
        thrust::distance(in_begin, in_end) >= NUM_UNICODE_ESC_SEQ_CHARS &&
        *in_begin == backslash_char && *thrust::next(in_begin) == 'u') {
      // Try to parse hex value following the '\' and 'u' characters from what may be a UTF16 low
      // surrogate
      hex_low_val = parse_unicode_hex(thrust::next(in_begin, 2));
    }

    // This is indeed a UTF16 surrogate pair
    if (hex_val >= UTF16_HIGH_SURROGATE_BEGIN && hex_val < UTF16_HIGH_SURROGATE_END &&
        hex_low_val >= UTF16_LOW_SURROGATE_BEGIN && hex_low_val < UTF16_LOW_SURROGATE_END) {
      // Skip over the second \uXXXX sequence
      thrust::advance(in_begin, NUM_UNICODE_ESC_SEQ_CHARS);

      // Compute UTF16-encoded code point
      uint32_t unicode_code_point = 0x10000 + ((hex_val - UTF16_HIGH_SURROGATE_BEGIN) << 10) +
                                    (hex_low_val - UTF16_LOW_SURROGATE_BEGIN);
      auto utf8_chars = strings::detail::codepoint_to_utf8(unicode_code_point);
      bytes += write_utf8_char(utf8_chars, d_buffer);
    } else {
      // Just a single \uXXXX sequence
      auto utf8_chars = strings::detail::codepoint_to_utf8(hex_val);
      bytes += write_utf8_char(utf8_chars, d_buffer);
    }
  }

  // The last character of the input is a backslash -> "fail"/null for this item
  if (is_prev_char_escape) { return {bytes, data_casting_result::PARSING_FAILURE}; }
  return {bytes, data_casting_result::PARSING_SUCCESS};
}

/**
 * @brief Data structure to hold 1 bit per thread with previous `UNICODE_LOOK_BACK` bits stored in a
 * warp.
 *
 * @tparam num_warps number of warps in the block
 */
template <unsigned num_warps>
struct bitfield_warp {
  static constexpr auto UNICODE_LOOK_BACK{5};
  // 5 because for skipping unicode hex chars, look back up to 5 chars are needed.
  // 5+32 for each warp.
  bool is_slash[num_warps][UNICODE_LOOK_BACK + cudf::detail::warp_size];

  /// Sets all bits to 0
  __device__ void reset(unsigned warp_id)
  {
    if (threadIdx.x % cudf::detail::warp_size < UNICODE_LOOK_BACK) {
      is_slash[warp_id][threadIdx.x % cudf::detail::warp_size] = 0;
    }
    is_slash[warp_id][threadIdx.x % cudf::detail::warp_size + UNICODE_LOOK_BACK] = 0;
  }

  /// Shifts UNICODE_LOOK_BACK bits to the left to hold the previous UNICODE_LOOK_BACK bits
  __device__ void shift(unsigned warp_id)
  {
    if (threadIdx.x % cudf::detail::warp_size < UNICODE_LOOK_BACK)
      is_slash[warp_id][threadIdx.x % cudf::detail::warp_size] =
        is_slash[warp_id][cudf::detail::warp_size + threadIdx.x % cudf::detail::warp_size];
    __syncwarp();
  }

  /// Each thread in a warp sets its own bit.
  __device__ void set_bits(unsigned warp_id, bool is_escaping_backslash)
  {
    is_slash[warp_id][UNICODE_LOOK_BACK + threadIdx.x % cudf::detail::warp_size] =
      is_escaping_backslash;
    __syncwarp();
  }

  /// Each thread in a warp gets the requested bit.
  __device__ bool get_bit(unsigned warp_id, int bit_index)
  {
    return is_slash[warp_id][UNICODE_LOOK_BACK + bit_index];
  }
};

/**
 * @brief Data structure to hold 1 bit per thread with previous `UNICODE_LOOK_BACK` bits stored in a
 * block.
 *
 * @tparam num_warps number of warps in the block
 */
template <unsigned num_warps>
struct bitfield_block {
  static constexpr auto UNICODE_LOOK_BACK{5};
  // 5 because for skipping unicode hex chars, look back up to 5 chars are needed.
  // 5 + num_warps*32 for entire block
  bool is_slash[UNICODE_LOOK_BACK + num_warps * cudf::detail::warp_size];

  /// Sets all bits to 0
  __device__ void reset(unsigned warp_id)
  {
    if (threadIdx.x < UNICODE_LOOK_BACK) { is_slash[threadIdx.x] = 0; }
    is_slash[threadIdx.x + UNICODE_LOOK_BACK] = 0;
  }

  /// Shifts UNICODE_LOOK_BACK bits to the left to hold the previous UNICODE_LOOK_BACK bits
  __device__ void shift(unsigned warp_id)
  {
    if (threadIdx.x < UNICODE_LOOK_BACK)
      is_slash[threadIdx.x] = is_slash[num_warps * cudf::detail::warp_size + threadIdx.x];
    __syncthreads();
  }

  /// Each thread in a block sets its own bit.
  __device__ void set_bits(unsigned warp_id, bool is_escaping_backslash)
  {
    is_slash[UNICODE_LOOK_BACK + threadIdx.x] = is_escaping_backslash;
    __syncthreads();
  }

  /// Each thread in a block gets the requested bit.
  __device__ bool get_bit(unsigned warp_id, int bit_index)
  {
    return is_slash[UNICODE_LOOK_BACK + bit_index];
  }
};

// Algorithm: warp/block parallel version of string_parse and process_string()
// Decoding character classes (u8, u16, \*, *):
// character      count: input->output
// \uXXXX         6->2/3/4
// \uXXXX\uXXXX  12->2/3/4
// \"             2->1
// *              1->1
//
// ERROR conditions. (all collaborating threads quit)
// c=='\' & curr_idx == end_idx-1;
// [c-1]=='\' &  get_escape[c]==NEC
// [c-1]=='\' &  [c]=='u' & end_idx-curr_idx < UNICODE_HEX_DIGIT_COUNT
// [c-1]=='\' &  [c]=='u' & end_idx-curr_idx >= UNICODE_HEX_DIGIT_COUNT && non-hex
//
// skip conditions. (current thread skips this char, no output)
// c=='\' skip. (Escaping char only)
// [c-2]=='\' && [c-1]=='u' for [2,1], [3,2] [4,5], [5, 6], skip.
//
// write conditions. (write to d_buffer)
// [c-1]!='\' &  [c]!='\' write [c]
// [c-1]!='\' &  [c]=='\' skip (already covered in skip conditions)
// [c-1]=='\' &  [c]!=NEC && [c]!=UNICODE_SEQ, write [c]
// [c-1]=='\' &  [c]=='u' & end_idx-curr_idx >= UNICODE_HEX_DIGIT_COUNT && hex, DECODE
// [c+1:4]=curr_hex_val
//        // if [c+5]=='\' & [c+6]=='u' & end_idx-curr_idx >= UNICODE_HEX_DIGIT_COUNT &&
//        hex,DECODE [c+7:4]=next_hex_val
//        // if [c-7]=='\' & [c-6]=='u' & end_idx-curr_idx >= UNICODE_HEX_DIGIT_COUNT &&
//        hex,DECODE [c-5:4]=prev_hex_val prev_hex_val, curr_hex_val, next_hex_val
//        // if prev_hex_val in high, curr_hex_val in low, skip.
//        // if curr_hex_val in high, next_hex_val in low, write [u16]
// if curr_hex_val not in high, write [u8]
// before writing, find num of output characters per threads,
// then do intra-warp/intra-block scan for out_idx
// propagate offset from next iteration to carry forward.
// Uses 1 warp per string or 1 block per string

/**
 * @brief Warp/Block parallel version of string_parse functor
 *
 * @tparam is_warp True if 1 warp per string, False if 1 block per string
 * @tparam num_warps Number of warps per block
 * @tparam str_tuple_it Iterator type for tuple with string pointer and its length
 * @param str_tuples iterator of tuple with string pointer and its length
 * @param total_out_strings Number of string rows to be processed
 * @param str_counter Counter to keep track of processed number of strings
 * @param null_mask Null mask
 * @param null_count_data pointer to store null count
 * @param options Settings for controlling string processing behavior
 * @param d_offsets Offsets to identify where to store the results for each string
 * @param d_chars Character array to store the characters of strings
 */
template <bool is_warp, size_type num_warps, typename str_tuple_it>
__global__ void parse_fn_string_parallel(str_tuple_it str_tuples,
                                         size_type total_out_strings,
                                         size_type* str_counter,
                                         bitmask_type* null_mask,
                                         size_type* null_count_data,
                                         cudf::io::parse_options_view const options,
                                         size_type* d_offsets,
                                         char* d_chars)
{
  constexpr auto BLOCK_SIZE =
    is_warp ? cudf::detail::warp_size : cudf::detail::warp_size * num_warps;
  size_type lane = is_warp ? (threadIdx.x % BLOCK_SIZE) : threadIdx.x;

  // get 1-string index per warp/block
  auto get_next_string = [&]() {
    if constexpr (is_warp) {
      size_type istring;
      if (lane == 0) { istring = atomicAdd(str_counter, 1); }
      return __shfl_sync(0xffffffff, istring, 0);
    } else {
      // Ensure lane 0 doesn't update istring before all threads have read the previous iteration's
      // istring value
      __syncthreads();
      __shared__ size_type istring;
      if (lane == 0) { istring = atomicAdd(str_counter, 1); }
      __syncthreads();
      return istring;
    }
  };
  // grid-stride loop.
  for (size_type istring = get_next_string(); istring < total_out_strings;
       istring           = get_next_string()) {
    // skip nulls
    if (null_mask != nullptr && not bit_is_set(null_mask, istring)) {
      if (!d_chars && lane == 0) d_offsets[istring] = 0;
      continue;  // gride-stride return;
    }

    auto in_begin           = str_tuples[istring].first;
    auto in_end             = in_begin + str_tuples[istring].second;
    auto const num_in_chars = str_tuples[istring].second;
    if constexpr (is_warp) {
      if (num_in_chars <= SINGLE_THREAD_THRESHOLD or num_in_chars > WARP_THRESHOLD) continue;
    } else {
      if (num_in_chars <= WARP_THRESHOLD) continue;
    }

    // Check if the value corresponds to the null literal
    if (!d_chars) {
      auto const is_null_literal = serialized_trie_contains(
        options.trie_na, {in_begin, static_cast<std::size_t>(num_in_chars)});
      if (is_null_literal && null_mask != nullptr) {
        if (lane == 0) {
          clear_bit(null_mask, istring);
          atomicAdd(null_count_data, 1);
          if (!d_chars) d_offsets[istring] = 0;
        }
        continue;  // gride-stride return;
      }
    }
    // String values are indicated by keeping the quote character
    bool const is_string_value =
      num_in_chars >= 2LL &&
      (options.quotechar == '\0' ||
       (*in_begin == options.quotechar) && (*thrust::prev(in_end) == options.quotechar));
    char* d_buffer = d_chars ? d_chars + d_offsets[istring] : nullptr;

    // Copy literal/numeric value
    if (not is_string_value) {
      if (!d_chars) {
        if (lane == 0) { d_offsets[istring] = in_end - in_begin; }
      } else {
        for (thread_index_type char_index = lane; char_index < (in_end - in_begin);
             char_index += BLOCK_SIZE) {
          d_buffer[char_index] = in_begin[char_index];
        }
      }
      continue;  // gride-stride return;
    }

    // Exclude beginning and ending quote chars from string range
    if (!options.keepquotes) {
      ++in_begin;
      --in_end;
    }
    // warp-parallelized or block-parallelized process_string()

    auto is_hex = [](auto ch) {
      return (ch >= '0' && ch <= '9') || (ch >= 'A' && ch <= 'F') || (ch >= 'a' && ch <= 'f');
    };

    // for backslash scan calculation: is_previous_escaping_backslash
    [[maybe_unused]] auto warp_id = threadIdx.x / cudf::detail::warp_size;
    bool init_state_reg;
    __shared__ bool init_state_shared;
    size_type last_offset_reg;
    __shared__ size_type last_offset_shared;
    bool& init_state(is_warp ? init_state_reg : init_state_shared);
    size_type& last_offset(is_warp ? last_offset_reg : last_offset_shared);
    if (is_warp || lane == 0) {
      init_state  = false;
      last_offset = 0;
    }
    using bitfield =
      std::conditional_t<is_warp, bitfield_warp<num_warps>, bitfield_block<num_warps>>;
    __shared__ bitfield is_slash;
    is_slash.reset(warp_id);
    __syncthreads();
    // 0-31, 32-63, ... i*32-n.
    // entire warp executes but with mask.
    for (thread_index_type char_index = lane;
         char_index < cudf::util::round_up_safe(in_end - in_begin, static_cast<long>(BLOCK_SIZE));
         char_index += BLOCK_SIZE) {
      bool const is_within_bounds = char_index < (in_end - in_begin);
      auto const MASK   = is_warp ? __ballot_sync(0xffffffff, is_within_bounds) : 0xffffffff;
      auto const c      = is_within_bounds ? in_begin[char_index] : '\0';
      auto const prev_c = (char_index > 0 and is_within_bounds) ? in_begin[char_index - 1] : '\0';
      auto const escaped_char = get_escape_char(c);

      bool is_escaping_backslash{false};
      [[maybe_unused]] bool is_prev_escaping_backslash{false};
      // To check current is backslash by checking if previous is backslash.
      // curr = !prev & c=='\\'
      // So, scan is required from beginning of string.
      // State table approach (intra-warp FST) (intra-block FST)
      // 2 states: Not-Slash(NS), Slash(S).
      // prev  /   *
      // NS    S  NS
      //  S   NS  NS
      // After inclusive scan, all current S states translate to escaping backslash.
      // All escaping backslash should be skipped.

      struct state_table {
        // using bit fields instead of state[2]
        bool state0 : 1;
        bool state1 : 1;
        bool inline __device__ get(bool init_state) const { return init_state ? state1 : state0; }
      };
      state_table curr{is_within_bounds && c == '\\', false};  // state transition vector.
      auto composite_op = [](state_table op1, state_table op2) {
        // equivalent of state_table{op2.state[op1.state[0]], op2.state[op1.state[1]]};
        return state_table{op1.state0 ? op2.state1 : op2.state0,
                           op1.state1 ? op2.state1 : op2.state0};
      };
      state_table scanned;
      // inclusive scan of escaping backslashes
      if constexpr (is_warp) {
        using SlashScan = hipcub::WarpScan<state_table>;
        __shared__ typename SlashScan::TempStorage temp_slash[num_warps];
        SlashScan(temp_slash[warp_id]).InclusiveScan(curr, scanned, composite_op);
        is_escaping_backslash = scanned.get(init_state);
        init_state            = __shfl_sync(MASK, is_escaping_backslash, BLOCK_SIZE - 1);
        __syncwarp();
        is_slash.shift(warp_id);
        is_slash.set_bits(warp_id, is_escaping_backslash);
        is_prev_escaping_backslash = is_slash.get_bit(warp_id, lane - 1);
      } else {
        using SlashScan = hipcub::BlockScan<state_table, BLOCK_SIZE>;
        __shared__ typename SlashScan::TempStorage temp_slash;
        SlashScan(temp_slash).InclusiveScan(curr, scanned, composite_op);
        is_escaping_backslash = scanned.get(init_state);
        __syncthreads();
        if (threadIdx.x == BLOCK_SIZE - 1) init_state = is_escaping_backslash;
        __syncthreads();
        is_slash.shift(warp_id);
        is_slash.set_bits(warp_id, is_escaping_backslash);
        is_prev_escaping_backslash = is_slash.get_bit(warp_id, lane - 1);
        // There is another __syncthreads() at the end of for-loop.
      }

      // String with parsing errors are made as null
      bool error = false;
      if (is_within_bounds) {
        // curr=='\' and end, or prev=='\' and curr=='u' and end-curr < UNICODE_HEX_DIGIT_COUNT
        // or prev=='\' and curr=='u' and end-curr >= UNICODE_HEX_DIGIT_COUNT and any non-hex
        error |= (is_escaping_backslash /*c == '\\'*/ && char_index == (in_end - in_begin) - 1);
        error |= (is_prev_escaping_backslash && escaped_char == NON_ESCAPE_CHAR);
        error |= (is_prev_escaping_backslash && c == 'u' &&
                  ((in_begin + char_index + UNICODE_HEX_DIGIT_COUNT >= in_end) |
                   !is_hex(in_begin[char_index + 1]) | !is_hex(in_begin[char_index + 2]) |
                   !is_hex(in_begin[char_index + 3]) | !is_hex(in_begin[char_index + 4])));
      }
      // Make sure all threads have no errors before continuing
      if constexpr (is_warp) {
        error = __any_sync(MASK, error);
      } else {
        using ErrorReduce = hipcub::BlockReduce<bool, BLOCK_SIZE>;
        __shared__ typename ErrorReduce::TempStorage temp_storage_error;
        __shared__ bool error_reduced;
        error_reduced = ErrorReduce(temp_storage_error).Sum(error);  // TODO use hipcub::LogicalOR.
        // only valid in thread0, so shared memory is used for broadcast.
        __syncthreads();
        error = error_reduced;
      }
      // If any thread has an error, skip the rest of the string and make this string as null
      if (error) {
        if (!d_chars && lane == 0) {
          if (null_mask != nullptr) {
            clear_bit(null_mask, istring);
            atomicAdd(null_count_data, 1);
          }
          last_offset        = 0;
          d_offsets[istring] = 0;
        }
        if constexpr (!is_warp) { __syncthreads(); }
        break;  // gride-stride return;
      }

      // Skipping non-copied escaped characters
      bool skip = !is_within_bounds;  // false;
      // skip \ for \" \\ \/ \b \f \n \r \t \uXXXX
      skip |= is_escaping_backslash;
      if (is_within_bounds) {
        // skip X for each X in \uXXXX
        skip |=
          char_index >= 2 && is_slash.get_bit(warp_id, lane - 2) && in_begin[char_index - 1] == 'u';
        skip |=
          char_index >= 3 && is_slash.get_bit(warp_id, lane - 3) && in_begin[char_index - 2] == 'u';
        skip |=
          char_index >= 4 && is_slash.get_bit(warp_id, lane - 4) && in_begin[char_index - 3] == 'u';
        skip |=
          char_index >= 5 && is_slash.get_bit(warp_id, lane - 5) && in_begin[char_index - 4] == 'u';
      }
      int this_num_out = 0;
      cudf::char_utf8 write_char{};

      if (!skip) {
        // 1. Unescaped character
        if (!is_prev_escaping_backslash) {
          this_num_out = 1;
          // writes char directly for non-unicode
        } else {
          // 2. Escaped character
          if (escaped_char != UNICODE_SEQ) {
            this_num_out = 1;
            // writes char directly for non-unicode
          } else {
            // 3. Unicode
            // UTF8 \uXXXX
            auto hex_val     = parse_unicode_hex(in_begin + char_index + 1);
            auto hex_low_val = 0;
            // UTF16 \uXXXX\uXXXX
            // Note: no need for scanned_backslash below because we already know that
            // only '\u' check is enough.
            if (hex_val >= UTF16_HIGH_SURROGATE_BEGIN && hex_val < UTF16_HIGH_SURROGATE_END &&
                (in_begin + char_index + UNICODE_HEX_DIGIT_COUNT + NUM_UNICODE_ESC_SEQ_CHARS) <
                  in_end &&
                in_begin[char_index + NUM_UNICODE_ESC_SEQ_CHARS - 1] == '\\' &&
                in_begin[char_index + NUM_UNICODE_ESC_SEQ_CHARS] == 'u') {
              hex_low_val = parse_unicode_hex(in_begin + char_index + 1 + 6);
            }
            if (hex_val >= UTF16_HIGH_SURROGATE_BEGIN && hex_val < UTF16_HIGH_SURROGATE_END &&
                hex_low_val >= UTF16_LOW_SURROGATE_BEGIN && hex_low_val < UTF16_LOW_SURROGATE_END) {
              // Compute UTF16-encoded code point
              uint32_t unicode_code_point = 0x10000 +
                                            ((hex_val - UTF16_HIGH_SURROGATE_BEGIN) << 10) +
                                            (hex_low_val - UTF16_LOW_SURROGATE_BEGIN);
              write_char   = strings::detail::codepoint_to_utf8(unicode_code_point);
              this_num_out = strings::detail::bytes_in_char_utf8(write_char);
            } else {
              // if hex_val is high surrogate, ideally it should be parsing failure.
              // but skipping it as other parsers do this too.
              if (hex_val >= UTF16_LOW_SURROGATE_BEGIN && hex_val < UTF16_LOW_SURROGATE_END) {
                // Ideally this should be skipped if previous char is high surrogate.
                skip         = true;
                this_num_out = 0;
                write_char   = 0;
              } else {
                // if UTF8
                write_char   = strings::detail::codepoint_to_utf8(hex_val);
                this_num_out = strings::detail::bytes_in_char_utf8(write_char);
              }
            }
          }
        }
      }  // !skip end.
      {
        // compute offset to write output for each thread
        size_type offset;
        if constexpr (is_warp) {
          using OffsetScan = hipcub::WarpScan<size_type>;
          __shared__ typename OffsetScan::TempStorage temp_storage[num_warps];
          OffsetScan(temp_storage[warp_id]).ExclusiveSum(this_num_out, offset);
        } else {
          using OffsetScan = hipcub::BlockScan<size_type, BLOCK_SIZE>;
          __shared__ typename OffsetScan::TempStorage temp_storage;
          OffsetScan(temp_storage).ExclusiveSum(this_num_out, offset);
          __syncthreads();
        }
        offset += last_offset;
        // Write output
        if (d_chars && !skip) {
          auto const is_not_unicode = (!is_prev_escaping_backslash) || escaped_char != UNICODE_SEQ;
          if (is_not_unicode) {
            *(d_buffer + offset) = (!is_prev_escaping_backslash) ? c : escaped_char;
          } else {
            strings::detail::from_char_utf8(write_char, d_buffer + offset);
          }
        }
        offset += this_num_out;
        if constexpr (is_warp) {
          last_offset = __shfl_sync(0xffffffff, offset, BLOCK_SIZE - 1);
        } else {
          __syncthreads();
          if (threadIdx.x == BLOCK_SIZE - 1) last_offset = offset;
          __syncthreads();
        }
      }
    }  // char for-loop
    if (!d_chars && lane == 0) { d_offsets[istring] = last_offset; }
  }  // grid-stride for-loop
}

template <typename str_tuple_it>
struct string_parse {
  str_tuple_it str_tuples;
  bitmask_type* null_mask;
  size_type* null_count_data;
  cudf::io::parse_options_view const options;
  size_type* d_offsets{};
  char* d_chars{};

  __device__ void operator()(size_type idx)
  {
    if (null_mask != nullptr && not bit_is_set(null_mask, idx)) {
      if (!d_chars) d_offsets[idx] = 0;
      return;
    }
    auto const in_begin     = str_tuples[idx].first;
    auto const in_end       = in_begin + str_tuples[idx].second;
    auto const num_in_chars = str_tuples[idx].second;

    if (num_in_chars > SINGLE_THREAD_THRESHOLD) return;

    // Check if the value corresponds to the null literal
    if (!d_chars) {
      auto const is_null_literal = serialized_trie_contains(
        options.trie_na, {in_begin, static_cast<std::size_t>(num_in_chars)});
      if (is_null_literal && null_mask != nullptr) {
        clear_bit(null_mask, idx);
        atomicAdd(null_count_data, 1);
        if (!d_chars) d_offsets[idx] = 0;
        return;
      }
    }

    char* d_buffer        = d_chars ? d_chars + d_offsets[idx] : nullptr;
    auto str_process_info = process_string(in_begin, in_end, d_buffer, options);
    if (str_process_info.result != data_casting_result::PARSING_SUCCESS) {
      if (null_mask != nullptr) {
        clear_bit(null_mask, idx);
        atomicAdd(null_count_data, 1);
      }
      if (!d_chars) d_offsets[idx] = 0;
    } else {
      if (!d_chars) d_offsets[idx] = str_process_info.bytes;
    }
  }
};

template <typename SymbolT>
struct to_string_view_pair {
  SymbolT const* data;
  to_string_view_pair(SymbolT const* _data) : data(_data) {}
  __device__ auto operator()(thrust::tuple<size_type, size_type> ip)
  {
    return thrust::pair<char const*, std::size_t>{data + thrust::get<0>(ip),
                                                  static_cast<std::size_t>(thrust::get<1>(ip))};
  }
};

template <typename string_view_pair_it>
static std::unique_ptr<column> parse_string(string_view_pair_it str_tuples,
                                            size_type col_size,
                                            rmm::device_buffer&& null_mask,
                                            rmm::device_scalar<size_type>& d_null_count,
                                            cudf::io::parse_options_view const& options,
                                            rmm::cuda_stream_view stream,
                                            rmm::mr::device_memory_resource* mr)
{
  //  CUDF_FUNC_RANGE();

  auto const max_length = thrust::transform_reduce(
    rmm::exec_policy(stream),
    str_tuples,
    str_tuples + col_size,
    [] __device__(auto t) { return t.second; },
    size_type{0},
    thrust::maximum<size_type>{});

  auto offsets = cudf::make_numeric_column(
    data_type{type_to_id<size_type>()}, col_size + 1, cudf::mask_state::UNALLOCATED, stream, mr);
  auto d_offsets       = offsets->mutable_view().data<size_type>();
  auto null_count_data = d_null_count.data();

  auto single_thread_fn = string_parse<decltype(str_tuples)>{
    str_tuples, static_cast<bitmask_type*>(null_mask.data()), null_count_data, options, d_offsets};
  thrust::for_each_n(rmm::exec_policy(stream),
                     thrust::make_counting_iterator<size_type>(0),
                     col_size,
                     single_thread_fn);

  constexpr auto warps_per_block  = 8;
  constexpr int threads_per_block = cudf::detail::warp_size * warps_per_block;
  auto num_blocks                 = cudf::util::div_rounding_up_safe(col_size, warps_per_block);
  auto str_counter                = cudf::numeric_scalar(size_type{0}, true, stream);

  // TODO run these independent kernels in parallel streams.
  if (max_length > SINGLE_THREAD_THRESHOLD) {
    parse_fn_string_parallel<true, warps_per_block>
      <<<num_blocks, threads_per_block, 0, stream.value()>>>(
        str_tuples,
        col_size,
        str_counter.data(),
        static_cast<bitmask_type*>(null_mask.data()),
        null_count_data,
        options,
        d_offsets,
        nullptr);
  }

  if (max_length > WARP_THRESHOLD) {
    // for strings longer than WARP_THRESHOLD, 1 block per string
    str_counter.set_value(0, stream);
    parse_fn_string_parallel<false, warps_per_block>
      <<<num_blocks, threads_per_block, 0, stream.value()>>>(
        str_tuples,
        col_size,
        str_counter.data(),
        static_cast<bitmask_type*>(null_mask.data()),
        null_count_data,
        options,
        d_offsets,
        nullptr);
  }
  auto const bytes =
    cudf::detail::sizes_to_offsets(d_offsets, d_offsets + col_size + 1, d_offsets, stream);
  CUDF_EXPECTS(bytes <= std::numeric_limits<size_type>::max(),
               "Size of output exceeds the column size limit",
               std::overflow_error);

  // CHARS column
  std::unique_ptr<column> chars =
    strings::detail::create_chars_child_column(static_cast<size_type>(bytes), stream, mr);
  auto d_chars = chars->mutable_view().data<char>();

  single_thread_fn.d_chars = d_chars;
  thrust::for_each_n(rmm::exec_policy(stream),
                     thrust::make_counting_iterator<size_type>(0),
                     col_size,
                     single_thread_fn);

  if (max_length > SINGLE_THREAD_THRESHOLD) {
    str_counter.set_value(0, stream);
    parse_fn_string_parallel<true, warps_per_block>
      <<<num_blocks, threads_per_block, 0, stream.value()>>>(
        str_tuples,
        col_size,
        str_counter.data(),
        static_cast<bitmask_type*>(null_mask.data()),
        null_count_data,
        options,
        d_offsets,
        d_chars);
  }

  if (max_length > WARP_THRESHOLD) {
    str_counter.set_value(0, stream);
    // for strings longer than WARP_THRESHOLD, 1 block per string
    parse_fn_string_parallel<false, warps_per_block>
      <<<num_blocks, threads_per_block, 0, stream.value()>>>(
        str_tuples,
        col_size,
        str_counter.data(),
        static_cast<bitmask_type*>(null_mask.data()),
        null_count_data,
        options,
        d_offsets,
        d_chars);
  }

  return make_strings_column(col_size,
                             std::move(offsets),
                             std::move(chars),
                             d_null_count.value(stream),
                             std::move(null_mask));
}

std::unique_ptr<column> parse_data(
  const char* data,
  thrust::zip_iterator<thrust::tuple<const size_type*, const size_type*>> offset_length_begin,
  size_type col_size,
  data_type col_type,
  rmm::device_buffer&& null_mask,
  size_type null_count,
  cudf::io::parse_options_view const& options,
  rmm::cuda_stream_view stream,
  rmm::mr::device_memory_resource* mr)
{
  CUDF_FUNC_RANGE();

  if (col_size == 0) { return make_empty_column(col_type); }
  auto d_null_count    = rmm::device_scalar<size_type>(null_count, stream);
  auto null_count_data = d_null_count.data();

  // Prepare iterator that returns (string_ptr, string_length)-pairs needed by type conversion
  auto str_tuples = thrust::make_transform_iterator(offset_length_begin, to_string_view_pair{data});

  if (col_type == cudf::data_type{cudf::type_id::STRING}) {
    return parse_string(str_tuples,
                        col_size,
                        std::forward<rmm::device_buffer>(null_mask),
                        d_null_count,
                        options,
                        stream,
                        mr);
  }

  auto out_col =
    make_fixed_width_column(col_type, col_size, std::move(null_mask), null_count, stream, mr);
  auto output_dv_ptr = mutable_column_device_view::create(*out_col, stream);

  // use `ConvertFunctor` to convert non-string values
  thrust::for_each_n(
    rmm::exec_policy(stream),
    thrust::make_counting_iterator<size_type>(0),
    col_size,
    [str_tuples, col = *output_dv_ptr, options, col_type, null_count_data] __device__(
      size_type row) {
      if (col.is_null(row)) { return; }
      auto const in = str_tuples[row];

      auto const is_null_literal =
        serialized_trie_contains(options.trie_na, {in.first, static_cast<size_t>(in.second)});

      if (is_null_literal) {
        col.set_null(row);
        atomicAdd(null_count_data, 1);
        return;
      }

      // If this is a string value, remove quotes
      auto [in_begin, in_end] = trim_quotes(in.first, in.first + in.second, options.quotechar);

      auto const is_parsed = cudf::type_dispatcher(col_type,
                                                   ConvertFunctor{},
                                                   in_begin,
                                                   in_end,
                                                   col.data<char>(),
                                                   row,
                                                   col_type,
                                                   options,
                                                   false);
      if (not is_parsed) {
        col.set_null(row);
        atomicAdd(null_count_data, 1);
      }
    });

  out_col->set_null_count(d_null_count.value(stream));

  return out_col;
}

}  // namespace cudf::io::json::detail
