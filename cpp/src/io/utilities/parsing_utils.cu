#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
	 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

/**
 * @file parsing_utils.cu Utility functions for parsing plain-text files
 *
 */


#include "parsing_utils.cuh"

#include <hip/hip_runtime.h>

#include <vector>
#include <memory>
#include <iostream>

#include <utilities/error_utils.hpp>
#include <io/utilities/wrapper_utils.hpp>

#include <rmm/device_buffer.hpp>

// When processing the input in chunks, this is the maximum size of each chunk.
// Only one chunk is loaded on the GPU at a time, so this value is chosen to
// be small enough to fit on the GPU in most cases.
constexpr size_t max_chunk_bytes = 256*1024*1024; // 256MB

constexpr int bytes_per_find_thread = 64;

using pos_key_pair = thrust::pair<uint64_t,char>;

template <typename T>
constexpr T divCeil(T dividend, T divisor) noexcept { return (dividend + divisor - 1) / divisor; }

/**---------------------------------------------------------------------------*
 * @brief Sets the specified element of the array to the passed value
 *---------------------------------------------------------------------------**/
template<class T, class V>
__device__ __forceinline__
void setElement(T* array, gdf_size_type idx, const T& t, const V& v){
	array[idx] = t;
}

/**---------------------------------------------------------------------------*
 * @brief Sets the specified element of the array of pairs using the two passed
 * parameters.
 *---------------------------------------------------------------------------**/
template<class T, class V>
__device__ __forceinline__
void setElement(thrust::pair<T, V>* array, gdf_size_type idx, const T& t, const V& v) {
	array[idx] = {t, v};
}

/**---------------------------------------------------------------------------*
 * @brief Overloads the setElement() functions for void* arrays.
 * Does not do anything, indexing is not allowed with void* arrays.
 *---------------------------------------------------------------------------**/
template<class T, class V>
__device__ __forceinline__
void setElement(void* array, gdf_size_type idx, const T& t, const V& v) {
}

/**---------------------------------------------------------------------------*
 * @brief CUDA kernel that finds all occurrences of a character in the given 
 * character array. If the 'positions' parameter is not void*,
 * positions of all occurrences are stored in the output array.
 * 
 * @param[in] data Pointer to the input character array
 * @param[in] size Number of bytes in the input array
 * @param[in] offset Offset to add to the output positions
 * @param[in] key Character to find in the array
 * @param[in,out] count Pointer to the number of found occurrences
 * @param[out] positions Array containing the output positions
 * 
 * @return void
 *---------------------------------------------------------------------------**/
template<class T>
 __global__ 
 void countAndSetPositions(char *data, uint64_t size, uint64_t offset, const char key, gdf_size_type* count,
	T* positions) {

	// thread IDs range per block, so also need the block id
	const uint64_t tid = threadIdx.x + (blockDim.x * blockIdx.x);
	const uint64_t did = tid * bytes_per_find_thread;
	
	const char *raw = (data + did);

	const long byteToProcess = ((did + bytes_per_find_thread) < size) ?
									bytes_per_find_thread :
									(size - did);

	// Process the data
	for (long i = 0; i < byteToProcess; i++) {
		if (raw[i] == key) {
			const auto idx = atomicAdd(count, (gdf_size_type)1);
			setElement(positions, idx, did + offset + i, key);
		}
	}
}

/**---------------------------------------------------------------------------*
 * @brief Searches the input character array for each of characters in a set.
 * Sums up the number of occurrences. If the 'positions' parameter is not void*,
 * positions of all occurrences are stored in the output device array.
 * 
 * Does not load the entire file into the GPU memory at any time, so it can 
 * be used to parse large files. Output array needs to be preallocated.
 * 
 * @param[in] h_data Pointer to the input character array
 * @param[in] h_size Number of bytes in the input array
 * @param[in] keys Vector containing the keys to count in the buffer
 * @param[in] result_offset Offset to add to the output positions
 * @param[out] positions Array containing the output positions
 * 
 * @return gdf_size_type total number of occurrences
 *---------------------------------------------------------------------------**/
template<class T>
gdf_size_type findAllFromSet(const char *h_data, size_t h_size, const std::vector<char>& keys, uint64_t result_offset,
	T *positions) {

	rmm::device_buffer d_chunk(std::min(max_chunk_bytes, h_size));
	rmm::device_vector<gdf_size_type> d_count(1, 0);

	int block_size = 0;		// suggested thread count to use
	int min_grid_size = 0;	// minimum block count required
	CUDA_TRY(hipOccupancyMaxPotentialBlockSize(&min_grid_size, &block_size, countAndSetPositions<T>) );

	const size_t chunk_count = divCeil(h_size, max_chunk_bytes);
	for (size_t ci = 0; ci < chunk_count; ++ci) {
		const auto chunk_offset = ci * max_chunk_bytes;
		const auto h_chunk = h_data + chunk_offset;
		const int chunk_bytes = std::min((size_t)(h_size - ci * max_chunk_bytes), max_chunk_bytes);
		const auto chunk_bits = divCeil(chunk_bytes, bytes_per_find_thread);
		const int grid_size = divCeil(chunk_bits, block_size);

		// Copy chunk to device
		CUDA_TRY(hipMemcpyAsync(d_chunk.data(), h_chunk, chunk_bytes, hipMemcpyDefault));

		for (char key: keys) {
			countAndSetPositions<T> <<< grid_size, block_size >>> (
				static_cast<char *>(d_chunk.data()), chunk_bytes,
				chunk_offset + result_offset, key, d_count.data().get(), positions);
		}
	}

	return d_count[0];
}

/**---------------------------------------------------------------------------*
 * @brief Searches the input character array for each of characters in a set
 * and sums up the number of occurrences.
 *
 * Does not load the entire buffer into the GPU memory at any time, so it can 
 * be used with buffers of any size.
 *
 * @param[in] h_data Pointer to the data in host memory
 * @param[in] h_size Size of the input data, in bytes
 * @param[in] keys Vector containing the keys to count in the buffer
 *
 * @return gdf_size_type total number of occurrences
 *---------------------------------------------------------------------------**/
gdf_size_type countAllFromSet(const char *h_data, size_t h_size, const std::vector<char>& keys) {
	return findAllFromSet<void>(h_data, h_size, keys, 0, nullptr);
}

template gdf_size_type findAllFromSet<uint64_t>(const char *h_data, size_t h_size, const std::vector<char>& keys, uint64_t result_offset,
	uint64_t *positions);

template gdf_size_type findAllFromSet<pos_key_pair>(const char *h_data, size_t h_size, const std::vector<char>& keys, uint64_t result_offset,
	pos_key_pair *positions);

/**
 * @brief A class representing an array of partial sums, stored in the GPU memory.
 *
 * The object is a reference to the device memory,
 * it does not own the allocated buffer.
 **/
struct BlockSumArray {
		int16_t* d_sums = nullptr;	///< Array of partial sums
		uint64_t length = 0;		///< Length of the array
		uint64_t block_size = 1;	///< The number of elements aggregated into each partial sum

		BlockSumArray(uint64_t len, uint64_t bsize): length(len), block_size(bsize){}
		BlockSumArray() noexcept = default;
};

/**
 * @brief A class that stores a pyramid of aggregated sums, in the GPU memory.
 *
 * Pyramid levels are stored bottom to top; each level is aggregation_rate
 * times smaller than the previous one, rounded down.
 * Objects of this type own the allocated memory.
 **/
class BlockSumPyramid {
	const uint16_t aggregation_rate_ = 32;			///< Aggregation rate between each level of the pyramid
	thrust::host_vector<BlockSumArray> h_levels_;	///< Host: pyramid levels (lowest to highest)
	rmm::device_vector<BlockSumArray> d_levels_;	///< Device: pyramid levels (lowest to highest)

public:
	BlockSumPyramid(int input_count){
		// input parameter is the number of elements aggregated with this pyramid
		int prev_count = input_count;
		int prev_block_size = 1;
		while (prev_count >= aggregation_rate_) {
			// We round down when computing the level sizes. Thus, there may be some elements in the input
			// array that are outside of the pyramid (up to aggregation_rate_ - 1 elements).
			h_levels_.push_back(BlockSumArray(prev_count/aggregation_rate_, prev_block_size*aggregation_rate_));
			RMM_ALLOC(&h_levels_.back().d_sums, h_levels_.back().length*sizeof(int16_t), 0);
			prev_count = h_levels_.back().length;
			prev_block_size = h_levels_.back().block_size;
		}

		if (!h_levels_.empty()) {
			d_levels_ = h_levels_;
		}
	}

	auto operator[](int level_idx) const {return h_levels_[level_idx];}
	auto deviceGetLevels() const noexcept {return d_levels_.data().get();}
	auto getHeight() const noexcept {return h_levels_.size();}
	auto getAggregationRate() const {return aggregation_rate_;}

	// disable copying
	BlockSumPyramid(BlockSumPyramid&) = delete;
	BlockSumPyramid& operator=(BlockSumPyramid&) = delete;

	~BlockSumPyramid() {
		for (auto& level: h_levels_) {
			RMM_FREE(level.d_sums, 0);
		}
	}
};

/**---------------------------------------------------------------------------*
 * @brief CUDA kernel that aggregates bracket nesting levels for each block
 * in the input array.
 *
 * Each sum is the level difference between the first bracket in the block,
 * and the first bracket in the next block (if any). For example, "[[]]" = 0,
 * because all open brackets are closed. "[[]" = 1, because the one unmatched
 * open bracket would raise the level of all subsequent elements.
 * 
 * @param[in] brackets Array of brackets, in (offset, char) format
 * @param[in] bracket_count Number of brackets
 * @param[in] open_chars Array of characters to treat as open brackets
 * @param[in] close_chars Array of characters to treat as close brackets
 * @param[in] bracket_char_cnt Number of bracket character pairs
 * @param[in, out] sum_array Array of partial sums
 * 
 * @return void
 *---------------------------------------------------------------------------**/
__global__
void sumBracketsKernel(
	pos_key_pair* brackets, int bracket_count,
	const char* open_chars, const char* close_chars, int bracket_char_cnt,
	BlockSumArray sum_array) {
	const uint64_t sum_idx = threadIdx.x + (blockDim.x * blockIdx.x);
	const uint64_t first_in_idx = sum_idx * sum_array.block_size;

	if (sum_idx >= sum_array.length)
		return;

	int16_t sum = 0;
	for (uint64_t in_idx = first_in_idx; in_idx < first_in_idx + sum_array.block_size; ++in_idx) {
		for (int bchar_idx = 0; bchar_idx < bracket_char_cnt; ++bchar_idx) {
			if (brackets[in_idx].second == open_chars[bchar_idx]) {
				++sum; 
				break;
			}
			if (brackets[in_idx].second == close_chars[bchar_idx]) {
				--sum; 
				break;
			}
		}
	}
	sum_array.d_sums[sum_idx] = sum;
}

/**---------------------------------------------------------------------------*
 * @brief Wrapper around sumBracketsKernel
 *
 * @param[in] brackets Array of brackets, in (offset, char) format
 * @param[in] bracket_count Number of brackets
 * @param[in] open_chars Array of characters to treat as open brackets
 * @param[in] close_chars Array of characters to treat as close brackets
 * @param[in] bracket_char_cnt Number of bracket character pairs
 * @param[in, out] sum_array Array of partial sums
 * 
 * @return void
 *---------------------------------------------------------------------------**/
void sumBrackets(
	pos_key_pair* brackets, int bracket_count,
	char* open_chars, char* close_chars, int bracket_char_cnt,
	const BlockSumArray& sum_array) {
	int block_size = 0;
	int min_grid_size = 0;
	CUDA_TRY(hipOccupancyMaxPotentialBlockSize(&min_grid_size, &block_size,
		sumBracketsKernel));

	const int gridSize = divCeil(sum_array.length, static_cast<uint64_t>(block_size));

	sumBracketsKernel<<<gridSize, block_size>>>(
		brackets, bracket_count,
		open_chars, close_chars, bracket_char_cnt,
		sum_array);
	CUDA_TRY(hipGetLastError());
};

/**---------------------------------------------------------------------------*
 * @brief CUDA kernel that computes partial sums of the input elements
 * 
 * @param[in] elements Array of input elements to sum
 * @param[in, out] aggregate Array of partial sums
 * 
 * @return void
 *---------------------------------------------------------------------------**/
__global__
void aggregateSumKernel(BlockSumArray elements, BlockSumArray aggregate){
	const uint64_t aggregate_idx = threadIdx.x + (blockDim.x * blockIdx.x);
	const int aggregate_group_size = aggregate.block_size / elements.block_size;
	const uint64_t first_in_idx = aggregate_idx * aggregate_group_size;

	if (aggregate_idx >= aggregate.length)
		return;

	int16_t sum = 0;
	for (int in_idx = first_in_idx; in_idx < first_in_idx + aggregate_group_size; ++in_idx) {
		sum += elements.d_sums[in_idx];
	}

	aggregate.d_sums[aggregate_idx] = sum;
}

/**---------------------------------------------------------------------------*
 * @brief Wrapper around aggregateSumKernel
 * 
 * @param[in] elements Array of input elements to sum
 * @param[in, out] aggregate Array of partial sums
 * 
 * @return void
 *---------------------------------------------------------------------------**/
void aggregateSum(const BlockSumArray& elements, const BlockSumArray& aggregate){
	int block_size = 0;
	int min_grid_size = 0;
	CUDA_TRY(hipOccupancyMaxPotentialBlockSize(&min_grid_size, &block_size,
		aggregateSumKernel));

	const int grid_size = divCeil(aggregate.length, static_cast<uint64_t>(block_size));

	aggregateSumKernel<<<grid_size, block_size>>>(elements, aggregate);
	CUDA_TRY(hipGetLastError());
};

/**---------------------------------------------------------------------------*
 * @brief CUDA kernel that assigns levels to each bracket,
 * with 1 being the top level
 *
 * The algorithm uses the pyramid of partial sums to compute the levels
 * in parallel, in log(n) time per block of elements.
 * 
 * @param[in] brackets Array of brackets, in (offset, char) format
 * @param[in] count Number of brackets
 * @param[in] sum_pyramid Pyramid of aggregated partial sums, where 
 * higher levels aggregate more elements per block
 * @param[in] pyramid_height Number of levels in the sum_pyramid
 * @param[in] open_chars Array of characters to treat as open brackets
 * @param[in] close_chars Array of characters to treat as close brackets
 * @param[in] bracket_char_cnt Number of bracket character pairs
 * @param[out] levels Array of output levels, one per bracket
 * 
 * @return void
 *---------------------------------------------------------------------------**/
__global__
void assignLevelsKernel(
	const pos_key_pair* brackets, uint64_t count,
	const BlockSumArray* sum_pyramid, int pyramid_height,
	const char* open_chars, const char* close_chars, int bracket_char_cnt,
	int16_t* levels) {
	// Process the number of elements equal to the aggregation rate, if the pyramid is used
	// Process all elements otherwise
	const auto to_process = pyramid_height != 0 ? sum_pyramid[0].block_size : count;
	const uint64_t tid = threadIdx.x + (blockDim.x * blockIdx.x);
	const uint64_t first_bracket_idx = tid * to_process;

	if (first_bracket_idx >= count)
		return;

	// Find the total sum of levels before the current block
	int sum = 0;
	if (pyramid_height != 0) {
		const auto aggregation_rate = sum_pyramid[0].block_size;
		int level = pyramid_height - 1;
		int block_idx = 0;
		int offset = first_bracket_idx;
		while(offset) {
			// Look for the highest level that can be used with the current offset
			while(offset < sum_pyramid[level].block_size && level > 0) {
				--level; block_idx *= aggregation_rate;
			}
			// Add up the blocks in the current level while the offset is after/at the block end
			while(offset >= sum_pyramid[level].block_size) {
				offset -= sum_pyramid[level].block_size;
				sum += sum_pyramid[level].d_sums[block_idx];
				++block_idx;
			}
		}
	}
	// Assign levels, update current level based on the encountered brackets
	const auto last_bracket_idx = min(first_bracket_idx + to_process, count) - 1;
	for (uint64_t bracket_idx = first_bracket_idx; bracket_idx <= last_bracket_idx; ++bracket_idx){
		for (int bchar_idx = 0; bchar_idx < bracket_char_cnt; ++bchar_idx) {
			if (brackets[bracket_idx].second == open_chars[bchar_idx]) {
				levels[bracket_idx] = ++sum;
				break;
			}
			else if (brackets[bracket_idx].second == close_chars[bchar_idx]) {
				levels[bracket_idx] = sum--;
				break;
			}
		}
	}
}

/**---------------------------------------------------------------------------*
 * @brief Wrapper around assignLevelsKernel
 * 
 * @param[in] brackets Array of brackets, in (offset, char) format
 * @param[in] count Number of brackets
 * @param[in] sum_pyramid Pyramid of aggregated partial sums, where 
 * higher levels aggregate more elements per block
 * @param[in] pyramid_height Number of levels in the sum_pyramid
 * @param[in] open_chars Array of characters to treat as open brackets
 * @param[in] close_chars Array of characters to treat as close brackets
 * @param[in] bracket_char_cnt Number of bracket character pairs
 * @param[out] levels Array of outout levels
 * 
 * @return void
 *---------------------------------------------------------------------------**/
void assignLevels(pos_key_pair* brackets, uint64_t count,
	const BlockSumPyramid& sum_pyramid,
	char* open_chars, char* close_chars, int bracket_char_cnt,
	int16_t* levels) {
	int block_size = 0;
	int min_grid_size = 0;
	CUDA_TRY(hipOccupancyMaxPotentialBlockSize(&min_grid_size, &block_size,
		assignLevelsKernel));

	const int thread_cnt = divCeil(count, static_cast<uint64_t>(sum_pyramid.getAggregationRate()));
	const int grid_size = divCeil(thread_cnt, block_size);

	assignLevelsKernel<<<grid_size, block_size>>>(
		brackets, count,
		sum_pyramid.deviceGetLevels(), sum_pyramid.getHeight(),
		open_chars, close_chars, bracket_char_cnt,
		levels);
	CUDA_TRY(hipGetLastError());
};

/**---------------------------------------------------------------------------*
 * @brief Computes nested levels for each of the brackets in the input array
 * 
 * The input array of brackets is sorted before levels are computed.
 * The algorithms assumes well-formed input, i.e. brackets are correctly nested
 * and there are no brackets that should be ignored (e.g. qouted brackets)
 * Brackets at the top level are assigned level 1.
 * 
 * @param[in] brackets Device memory array of brackets, in (offset, key) format
 * @param[in] count Number of brackets
 * @param[in] open_chars string of characters to treat as open brackets
 * @param[in] close_chars string of characters to treat as close brackets
 * 
 * @return rmm::device_vector<int16_t> Device vector containing bracket levels
 *---------------------------------------------------------------------------**/
rmm::device_vector<int16_t> getBracketLevels(
	pos_key_pair* brackets, int count,
	const std::string& open_chars, const std::string& close_chars){
	// TODO: consider moving sort() out of this function
	thrust::sort(rmm::exec_policy()->on(0), brackets, brackets + count);

	// Total bracket level difference within each segment of brackets
	BlockSumPyramid aggregated_sums(count);
	
	CUDF_EXPECTS(open_chars.size() == close_chars.size(),
		"The number of open and close bracket characters must be equal.");

	// Copy the open/close chars to device
	rmm::device_buffer d_open_chars(open_chars.data(), open_chars.size());
	rmm::device_buffer d_close_chars(close_chars.data(), close_chars.size());

	if (aggregated_sums.getHeight() != 0) {
		sumBrackets(
			brackets, count, static_cast<char *>(d_open_chars.data()),
			static_cast<char *>(d_close_chars.data()), open_chars.size(),
			aggregated_sums[0]);
		for (size_t level_idx = 1; level_idx < aggregated_sums.getHeight(); ++level_idx)
			aggregateSum(aggregated_sums[level_idx - 1], aggregated_sums[level_idx]);
	}

	rmm::device_vector<int16_t> d_levels(count);
	assignLevels(
		brackets, count, aggregated_sums,
		static_cast<char *>(d_open_chars.data()),
		static_cast<char *>(d_close_chars.data()),
		open_chars.size(), d_levels.data().get());

	return std::move(d_levels);
}
