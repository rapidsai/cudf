#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "avro.h"
#include "avro_gpu.h"
#include "avro_reader_impl.hpp"

#include "io/comp/gpuinflate.h"

#include <rmm/device_buffer.hpp>

namespace cudf {
namespace io {
namespace avro {

#if 0
#define LOG_PRINTF(...) std::printf(__VA_ARGS__)
#else
#define LOG_PRINTF(...) (void)0
#endif

/**
 * @brief Function that translates Avro datatype to GDF dtype
 **/
gdf_dtype to_dtype(const avro::schema_entry *col) {
  switch (col->kind) {
    case avro::type_boolean:
      return GDF_BOOL8;
    case avro::type_int:
      return GDF_INT32;
    case avro::type_long:
      return GDF_INT64;
    case avro::type_float:
      return GDF_FLOAT32;
    case avro::type_double:
      return GDF_FLOAT64;
    case avro::type_bytes:
    case avro::type_string:
      return GDF_STRING;
    case avro::type_enum:
      return (!col->symbols.empty()) ? GDF_STRING : GDF_INT32;
    default:
      return GDF_invalid;
  }
}

/**
 * @brief A helper wrapper for Avro file metadata. Provides some additional
 * convenience methods for initializing and accessing the metadata and schema
 **/
class avro_metadata : public avro::file_metadata {
 public:
  explicit avro_metadata(datasource *const src) : source(src) {}

  /**
   * @brief Initializes the parser and filters down to a subset of rows
   *
   * @param[in,out] row_start Starting row of the selection
   * @param[in,out] row_count Total number of rows selected
   **/
  void init_and_select_rows(int &row_start, int &row_count) {
    const auto buffer = source->get_buffer(0, source->size());
    avro::container pod(buffer->data(), buffer->size());
    CUDF_EXPECTS(pod.parse(this, row_count, row_start),
                 "Cannot parse metadata");
    print_metadata();
    row_start = skip_rows;
    row_count = num_rows;
  }

  /**
   * @brief Filters and reduces down to a selection of columns
   *
   * @param[in] use_names List of column names to select
   *
   * @return List of column names
   **/
  auto select_columns(std::vector<std::string> use_names) {
    std::vector<std::pair<int, std::string>> selection;

    const auto num_avro_columns = static_cast<int>(columns.size());
    if (!use_names.empty()) {
      int index = 0;
      for (const auto &use_name : use_names) {
        for (int i = 0; i < num_avro_columns; ++i, ++index) {
          if (index >= num_avro_columns) {
            index = 0;
          }
          if (columns[index].name == use_name &&
              GDF_invalid !=
                  to_dtype(&schema[columns[index].schema_data_idx])) {
            selection.emplace_back(index, columns[index].name);
            index++;
            break;
          }
        }
      }
    } else {
      // Iterate backwards as fastavro returns from last-to-first?!
      for (int i = num_avro_columns - 1; i >= 0; --i) {
        const auto dtype = to_dtype(&schema[columns[i].schema_data_idx]);
        CUDF_EXPECTS(dtype != GDF_invalid, "Unsupported data type");
        selection.emplace_back(i, columns[i].name);
      }
    }
    CUDF_EXPECTS(selection.size() > 0, "Filtered out all columns");

    return selection;
  }

  void print_metadata() const {
    LOG_PRINTF("\n[+] Metadata:\n");
    LOG_PRINTF(" size = %zd\n", metadata_size);
    LOG_PRINTF(" codec = \"%s\"\n", codec.c_str());
    LOG_PRINTF(" sync marker = 0x%016" PRIx64 "%016" PRIx64 "\n",
               sync_marker[1], sync_marker[0]);
    LOG_PRINTF(" schema (%zd entries):\n", schema.size());
    for (size_t i = 0; i < schema.size(); ++i) {
      LOG_PRINTF(
          "  [%zd] num_children=%d, parent_idx=%d, type=%d, name=\"%s\"\n", i,
          schema[i].num_children, schema[i].parent_idx, schema[i].kind,
          schema[i].name.c_str());
    }
    LOG_PRINTF(" datablocks (%zd entries):\n", block_list.size());
    LOG_PRINTF(" num rows = %zd (max block size = %d, total_data_size = %zd)\n",
               num_rows, max_block_size, total_data_size);
    LOG_PRINTF(" num columns = %zd\n", columns.size());
    LOG_PRINTF(" user data entries = %zd\n", user_data.size());
    for (const auto &entry : user_data) {
      LOG_PRINTF("  key: %s, value: %s\n", entry.first.c_str(),
                 entry.second.c_str());
    }
  }

 private:
  datasource *const source;
};

reader::Impl::Impl(std::unique_ptr<datasource> source,
                   reader_options const &options)
    : source_(std::move(source)), columns_(options.columns) {

  // Open the source Avro dataset metadata
  md_ = std::make_unique<avro_metadata>(source_.get());
}

table reader::Impl::read(int skip_rows, int num_rows) {
  // Select and read partial metadata / schema within the subset of rows
  md_->init_and_select_rows(skip_rows, num_rows);

  // Select only columns required by the options
  selected_cols_ = md_->select_columns(columns_);
  if (selected_cols_.empty()) {
    return table();
  }

  // Initialize gdf_columns, but hold off on allocating storage space
  LOG_PRINTF("[+] Selected columns: %zd\n", selected_cols_.size());
  LOG_PRINTF("[+] Selected skip_rows: %d, num_rows: %d\n", skip_rows, num_rows);
  std::vector<gdf_column_wrapper> columns;
  for (const auto &col : selected_cols_) {
    auto &col_schema = md_->schema[md_->columns[col.first].schema_data_idx];

    columns.emplace_back(static_cast<gdf_size_type>(num_rows),
                         to_dtype(&col_schema),
                         gdf_dtype_extra_info{TIME_UNIT_NONE}, col.second);

    LOG_PRINTF(" %2zd: name=%s size=%zd type=%d data=%lx valid=%lx\n",
               columns.size() - 1, columns.back()->col_name,
               (size_t)columns.back()->size, columns.back()->dtype,
               (uint64_t)columns.back()->data, (uint64_t)columns.back()->valid);
  }

  if (md_->total_data_size > 0) {
    const auto buffer =
        source_->get_buffer(md_->block_list[0].offset, md_->total_data_size);
    rmm::device_buffer block_data(buffer->data(), align_size(buffer->size()));

    if (md_->codec != "" && md_->codec != "null") {
      auto decomp_block_data = decompress_data(block_data);
      block_data = std::move(decomp_block_data);
    } else {
      auto dst_ofs = md_->block_list[0].offset;
      for (size_t i = 0; i < md_->block_list.size(); i++) {
        md_->block_list[i].offset -= dst_ofs;
      }
    }

    size_t total_dictionary_entries = 0;
    size_t dictionary_data_size = 0;
    std::vector<std::pair<uint32_t, uint32_t>> dict(columns.size());
    for (size_t i = 0; i < columns.size(); ++i) {
      columns[i].allocate();
      size_t valid_bytes = columns[i]->size >> 3;
      size_t valid_size = gdf_valid_allocation_size(columns[i]->size);
      uint8_t *valid = reinterpret_cast<uint8_t *>(columns[i]->valid);
      CUDA_TRY(hipMemsetAsync(valid, -1, valid_bytes));
      if (columns[i]->size & 7) {
        CUDA_TRY(hipMemsetAsync(valid + valid_bytes, (1 << (columns[i]->size & 7)) - 1, 1));
        valid_bytes++;
      }
      if (valid_bytes < valid_size) {
        CUDA_TRY(hipMemsetAsync(valid + valid_bytes, 0, valid_size - valid_bytes));
      }
      auto col_idx = selected_cols_[i].first;
      auto &col_schema = md_->schema[md_->columns[col_idx].schema_data_idx];
      dict[i].first = static_cast<uint32_t>(total_dictionary_entries);
      dict[i].second = static_cast<uint32_t>(col_schema.symbols.size());
      total_dictionary_entries += dict[i].second;
      for (const auto &sym : col_schema.symbols) {
        dictionary_data_size += sym.length();
      }
    }

    hostdevice_vector<uint8_t> global_dictionary(total_dictionary_entries * sizeof(gpu::nvstrdesc_s) + dictionary_data_size);
    if (total_dictionary_entries > 0) {
      size_t dict_pos = total_dictionary_entries * sizeof(gpu::nvstrdesc_s);
      for (size_t i = 0; i < columns.size(); ++i) {
        auto col_idx = selected_cols_[i].first;
        auto &col_schema = md_->schema[md_->columns[col_idx].schema_data_idx];
        auto index = &(reinterpret_cast<gpu::nvstrdesc_s *>(global_dictionary.host_ptr()))[dict[i].first];
        for (size_t j = 0; j < dict[i].second; j++) {
          size_t len = col_schema.symbols[j].length();
          char *ptr = reinterpret_cast<char *>(global_dictionary.device_ptr() +
                                               dict_pos);
          index[j].ptr = ptr;
          index[j].count = len;
          memcpy(global_dictionary.host_ptr() + dict_pos,
                 col_schema.symbols[j].c_str(), len);
          dict_pos += len;
        }
      }
      CUDA_TRY(hipMemcpyAsync(
          global_dictionary.device_ptr(), global_dictionary.host_ptr(),
          global_dictionary.memory_size(), hipMemcpyHostToDevice));
    }

    // Write out columns
    decode_data(block_data, dict, global_dictionary, total_dictionary_entries,
                columns);

    // Perform any final column preparation (may reference decoded data)
    for (auto &column : columns) {
      column.finalize();
    }
  } else {
    for (auto &column : columns) {
      column.allocate();
      column.finalize();
    }
  }

  // Transfer ownership to raw pointer output arguments
  std::vector<gdf_column *> out_cols(columns.size());
  for (size_t i = 0; i < columns.size(); ++i) {
    out_cols[i] = columns[i].release();
  }

  return cudf::table(out_cols.data(), out_cols.size());
}

rmm::device_buffer reader::Impl::decompress_data(
    const rmm::device_buffer &comp_block_data) {
  size_t uncompressed_data_size = 0;
  hostdevice_vector<gpu_inflate_input_s> inflate_in(md_->block_list.size());
  hostdevice_vector<gpu_inflate_status_s> inflate_out(md_->block_list.size());

  if (md_->codec == "deflate") {
    // Guess an initial maximum uncompressed block size
    uint32_t initial_blk_len = (md_->max_block_size * 2 + 0xfff) & ~0xfff;
    uncompressed_data_size = initial_blk_len * md_->block_list.size();
    for (size_t i = 0; i < inflate_in.size(); ++i) {
      inflate_in[i].dstSize = initial_blk_len;
    }
  } else if (md_->codec == "snappy") {
    // Extract the uncompressed length from the snappy stream
    for (size_t i = 0; i < md_->block_list.size(); i++) {
      const auto buffer = source_->get_buffer(md_->block_list[i].offset, 4);
      const uint8_t *blk = buffer->data();
      uint32_t blk_len = blk[0];
      if (blk_len > 0x7f) {
        blk_len = (blk_len & 0x7f) | (blk[1] << 7);
        if (blk_len > 0x3fff) {
          blk_len = (blk_len & 0x3fff) | (blk[2] << 14);
          if (blk_len > 0x1fffff) {
            blk_len = (blk_len & 0x1fffff) | (blk[3] << 21);
          }
        }
      }
      inflate_in[i].dstSize = blk_len;
      uncompressed_data_size += blk_len;
    }
  } else {
    CUDF_FAIL("Unsupported compression codec\n");
  }

  rmm::device_buffer decomp_block_data(uncompressed_data_size);

  const auto base_offset = md_->block_list[0].offset;
  for (size_t i = 0, dst_pos = 0; i < md_->block_list.size(); i++) {
    const auto src_pos = md_->block_list[i].offset - base_offset;

    inflate_in[i].srcDevice =
        static_cast<const uint8_t *>(comp_block_data.data()) + src_pos;
    inflate_in[i].srcSize = md_->block_list[i].size;
    inflate_in[i].dstDevice =
        static_cast<uint8_t *>(decomp_block_data.data()) + dst_pos;

    // Update blocks offsets & sizes to refer to uncompressed data
    md_->block_list[i].offset = dst_pos;
    md_->block_list[i].size = static_cast<uint32_t>(inflate_in[i].dstSize);
    dst_pos += md_->block_list[i].size;
  }

  for (int loop_cnt = 0; loop_cnt < 2; loop_cnt++) {
    CUDA_TRY(hipMemcpyAsync(inflate_in.device_ptr(), inflate_in.host_ptr(),
                             inflate_in.memory_size(), hipMemcpyHostToDevice));
    CUDA_TRY(hipMemsetAsync(inflate_out.device_ptr(), 0,
                             inflate_out.memory_size()));
    if (md_->codec == "deflate") {
      CUDA_TRY(gpuinflate(inflate_in.device_ptr(), inflate_out.device_ptr(),
                          inflate_in.size(), 0));
    } else if (md_->codec == "snappy") {
      CUDA_TRY(gpu_unsnap(inflate_in.device_ptr(), inflate_out.device_ptr(),
                          inflate_in.size()));
    } else {
      CUDF_FAIL("Unsupported compression codec\n");
    }
    CUDA_TRY(hipMemcpyAsync(inflate_out.host_ptr(), inflate_out.device_ptr(),
                             inflate_out.memory_size(),
                             hipMemcpyDeviceToHost));
    CUDA_TRY(hipStreamSynchronize(0));

    // Check if larger output is required, as it's not known ahead of time
    if (md_->codec == "deflate" && !loop_cnt) {
      size_t actual_uncompressed_size = 0;
      for (size_t i = 0; i < md_->block_list.size(); i++) {
        // If error status is 1 (buffer too small), the `bytes_written` field
        // is actually contains the uncompressed data size
        if (inflate_out[i].status == 1 &&
            inflate_out[i].bytes_written > inflate_in[i].dstSize) {
          inflate_in[i].dstSize = inflate_out[i].bytes_written;
        }
        actual_uncompressed_size += inflate_in[i].dstSize;
      }
      if (actual_uncompressed_size > uncompressed_data_size) {
        decomp_block_data.resize(actual_uncompressed_size);
        for (size_t i = 0, dst_pos = 0; i < md_->block_list.size(); i++) {
          auto dst_base = static_cast<uint8_t *>(decomp_block_data.data());
          inflate_in[i].dstDevice = dst_base + dst_pos;

          md_->block_list[i].offset = dst_pos;
          md_->block_list[i].size = static_cast<uint32_t>(inflate_in[i].dstSize);
          dst_pos += md_->block_list[i].size;
        }
      } else {
        break;
      }
    } else {
      break;
    }
  }

  return decomp_block_data;
}

void reader::Impl::decode_data(
    const rmm::device_buffer &block_data,
    const std::vector<std::pair<uint32_t, uint32_t>> &dict,
    const hostdevice_vector<uint8_t> &global_dictionary,
    size_t total_dictionary_entries,
    const std::vector<gdf_column_wrapper> &columns) {
  // Build gpu schema
  hostdevice_vector<gpu::schemadesc_s> schema_desc(md_->schema.size());
  uint32_t min_row_data_size = 0;
  int skip_field_cnt = 0;
  for (size_t i = 0; i < md_->schema.size(); i++) {
    type_kind_e kind = md_->schema[i].kind;
    if (skip_field_cnt != 0) {
      // Exclude union members from min_row_data_size
      skip_field_cnt += md_->schema[i].num_children - 1;
    } else {
      switch (kind) {
        case type_union:
          skip_field_cnt = md_->schema[i].num_children;
          // fall through
        case type_boolean:
        case type_int:
        case type_long:
        case type_bytes:
        case type_string:
        case type_enum:
          min_row_data_size += 1;
          break;
        case type_float:
          min_row_data_size += 4;
          break;
        case type_double:
          min_row_data_size += 8;
          break;
        default:
          break;
      }
    }
    if (kind == type_enum && !md_->schema[i].symbols.size()) {
      kind = type_int;
    }
    schema_desc[i].kind = kind;
    schema_desc[i].count = (kind == type_enum) ? 0 : (uint32_t)md_->schema[i].num_children;
    schema_desc[i].dataptr = nullptr;
    CUDF_EXPECTS(kind != type_union || md_->schema[i].num_children < 2 ||
                     (md_->schema[i].num_children == 2 &&
                      (md_->schema[i + 1].kind == type_null ||
                       md_->schema[i + 2].kind == type_null)),
                 "Union with non-null type not currently supported");
  }
  std::vector<void*> valid_alias(columns.size(), nullptr);
  for (size_t i = 0; i < columns.size(); i++) {
    auto col_idx = selected_cols_[i].first;
    int schema_data_idx = md_->columns[col_idx].schema_data_idx;
    int schema_null_idx = md_->columns[col_idx].schema_null_idx;
    schema_desc[schema_data_idx].dataptr = columns[i]->data;
    if (schema_null_idx >= 0) {
      if (!schema_desc[schema_null_idx].dataptr) {
        schema_desc[schema_null_idx].dataptr = columns[i]->valid;
      } else {
        valid_alias[i] = schema_desc[schema_null_idx].dataptr;
      }
    }
    if (md_->schema[schema_data_idx].kind == type_enum) {
      schema_desc[schema_data_idx].count = dict[i].first;
    }
  }
  rmm::device_buffer block_list(md_->block_list.data(),
                                md_->block_list.size() * sizeof(block_desc_s));
  CUDA_TRY(hipMemcpyAsync(schema_desc.device_ptr(), schema_desc.host_ptr(),
                           schema_desc.memory_size(), hipMemcpyHostToDevice));

  CUDA_TRY(DecodeAvroColumnData(
      static_cast<block_desc_s *>(block_list.data()), schema_desc.device_ptr(),
      reinterpret_cast<gpu::nvstrdesc_s *>(global_dictionary.device_ptr()),
      static_cast<const uint8_t *>(block_data.data()),
      static_cast<uint32_t>(block_list.size()),
      static_cast<uint32_t>(schema_desc.size()),
      static_cast<uint32_t>(total_dictionary_entries), md_->num_rows,
      md_->skip_rows, min_row_data_size, 0));

  // Copy valid bits that are shared between columns
  for (size_t i = 0; i < columns.size(); i++) {
    if (valid_alias[i] != nullptr) {
      CUDA_TRY(hipMemcpyAsync(columns[i]->valid, valid_alias[i],
                               gdf_valid_allocation_size(columns[i]->size),
                               hipMemcpyHostToDevice));
    }
  }
  CUDA_TRY(hipMemcpyAsync(schema_desc.host_ptr(), schema_desc.device_ptr(),
                           schema_desc.memory_size(), hipMemcpyDeviceToHost));
  CUDA_TRY(hipStreamSynchronize(0));
  for (size_t i = 0; i < columns.size(); i++) {
    const auto col_idx = selected_cols_[i].first;
    const auto schema_null_idx = md_->columns[col_idx].schema_null_idx;
    columns[i]->null_count =
        (schema_null_idx >= 0) ? schema_desc[schema_null_idx].count : 0;
  }
}

reader::reader(std::string filepath, reader_options const &options)
    : impl_(std::make_unique<Impl>(datasource::create(filepath), options)) {}

reader::reader(const char *buffer, size_t length, reader_options const &options)
    : impl_(std::make_unique<Impl>(datasource::create(buffer, length),
                                   options)) {}

reader::reader(std::shared_ptr<arrow::io::RandomAccessFile> file,
               reader_options const &options)
    : impl_(std::make_unique<Impl>(datasource::create(file), options)) {}

table reader::read_all() { return impl_->read(0, -1); }

table reader::read_rows(size_t skip_rows, size_t num_rows) {
  return impl_->read(skip_rows,
                     (num_rows != 0) ? static_cast<int>(num_rows) : -1);
}

reader::~reader() = default;

}  // namespace avro
}  // namespace io
}  // namespace cudf
