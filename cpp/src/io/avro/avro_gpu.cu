#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019-2020, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */
#include "avro_gpu.h"

#include <io/utilities/block_utils.cuh>

using cudf::detail::device_span;

namespace cudf {
namespace io {
namespace avro {
namespace gpu {
#define NWARPS 16
#define MAX_SHARED_SCHEMA_LEN 1000

/*
 * Avro varint encoding - see
 * https://avro.apache.org/docs/1.2.0/spec.html#binary_encoding
 */
static inline int64_t __device__ avro_decode_zigzag_varint(const uint8_t *&cur, const uint8_t *end)
{
  uint64_t u = 0;
  if (cur < end) {
    u = *cur++;
    if (u > 0x7f) {
      uint64_t scale = 128;
      u &= 0x7f;
      while (cur < end) {
        uint32_t c = *cur++;
        u += (c & 0x7f) * scale;
        scale <<= 7;
        if (c < 0x80) break;
      }
    }
  }
  return (int64_t)((u >> 1u) ^ -(int64_t)(u & 1));
}

/**
 * @brief Decode a row of values given an avro schema
 *
 * @param[in] schema Schema description
 * @param[in] schema_g Global schema in device mem
 * @param[in] schema_len Number of schema entries
 * @param[in] row Current row
 * @param[in] max_rows Total number of rows
 * @param[in] cur Current input data pointer
 * @param[in] end End of input data
 * @param[in] global_Dictionary Global dictionary entries
 *
 * @return data pointer at the end of the row (start of next row)
 *
 **/
static const uint8_t *__device__ avro_decode_row(const schemadesc_s *schema,
                                                 schemadesc_s *schema_g,
                                                 uint32_t schema_len,
                                                 size_t row,
                                                 size_t max_rows,
                                                 const uint8_t *cur,
                                                 const uint8_t *end,
                                                 device_span<nvstrdesc_s> global_dictionary)
{
  uint32_t array_start = 0, array_repeat_count = 0;
  int array_children = 0;
  for (uint32_t i = 0; i < schema_len;) {
    uint32_t kind = schema[i].kind;
    int skip      = 0;

    if (kind == type_union) {
      int skip_after;
      if (cur >= end) break;
      skip       = (*cur++) >> 1;  // NOTE: Assumes 1-byte union member
      skip_after = schema[i].count - skip - 1;
      ++i;
      while (skip > 0 && i < schema_len) {
        if (schema[i].kind >= type_record) { skip += schema[i].count; }
        ++i;
        --skip;
      }
      if (i >= schema_len || skip_after < 0) break;
      kind = schema[i].kind;
      skip = skip_after;
    }

    void *dataptr = schema[i].dataptr;
    switch (kind) {
      case type_null:
        if (dataptr != nullptr && row < max_rows) {
          atomicAnd(static_cast<uint32_t *>(dataptr) + (row >> 5), ~(1 << (row & 0x1f)));
          atomicAdd(&schema_g[i].count, 1);
        }
        break;

      case type_int:
      case type_long:
      case type_bytes:
      case type_string:
      case type_enum: {
        int64_t v = avro_decode_zigzag_varint(cur, end);
        if (kind == type_int) {
          if (dataptr != nullptr && row < max_rows) {
            static_cast<int32_t *>(dataptr)[row] = static_cast<int32_t>(v);
          }
        } else if (kind == type_long) {
          if (dataptr != nullptr && row < max_rows) { static_cast<int64_t *>(dataptr)[row] = v; }
        } else {  // string or enum
          size_t count    = 0;
          const char *ptr = 0;
          if (kind == type_enum) {  // dictionary
            size_t idx = schema[i].count + v;
            if (idx < global_dictionary.size()) {
              ptr   = global_dictionary[idx].ptr;
              count = global_dictionary[idx].count;
            }
          } else if (v >= 0 && cur + v <= end) {  // string
            ptr   = reinterpret_cast<const char *>(cur);
            count = (size_t)v;
            cur += count;
          }
          if (dataptr != nullptr && row < max_rows) {
            static_cast<nvstrdesc_s *>(dataptr)[row].ptr   = ptr;
            static_cast<nvstrdesc_s *>(dataptr)[row].count = count;
          }
        }
      } break;

      case type_float:
        if (dataptr != nullptr && row < max_rows) {
          uint32_t v;
          if (cur + 3 < end) {
            v = unaligned_load32(cur);
            cur += 4;
          } else {
            v = 0;
          }
          static_cast<uint32_t *>(dataptr)[row] = v;
        } else {
          cur += 4;
        }
        break;

      case type_double:
        if (dataptr != nullptr && row < max_rows) {
          uint64_t v;
          if (cur + 7 < end) {
            v = unaligned_load64(cur);
            cur += 8;
          } else {
            v = 0;
          }
          static_cast<uint64_t *>(dataptr)[row] = v;
        } else {
          cur += 8;
        }
        break;

      case type_boolean:
        if (dataptr != nullptr && row < max_rows) {
          uint8_t v                            = (cur < end) ? *cur : 0;
          static_cast<uint8_t *>(dataptr)[row] = (v) ? 1 : 0;
        }
        cur++;
        break;

      case type_array: {
        int32_t array_block_count = avro_decode_zigzag_varint(cur, end);
        if (array_block_count < 0) {
          avro_decode_zigzag_varint(cur, end);  // block size in bytes, ignored
          array_block_count = -array_block_count;
        }
        array_start        = i;
        array_repeat_count = array_block_count;
        array_children     = 1;
        if (array_repeat_count == 0) {
          skip += schema[i].count;  // Should always be 1
        }
      } break;
    }
    if (array_repeat_count != 0) {
      array_children--;
      if (schema[i].kind >= type_record) { array_children += schema[i].count; }
    }
    i++;
    while (skip > 0 && i < schema_len) {
      if (schema[i].kind >= type_record) { skip += schema[i].count; }
      ++i;
      --skip;
    }
    // If within an array, check if we reached the last item
    if (array_repeat_count != 0 && array_children <= 0 && cur < end) {
      if (!--array_repeat_count) {
        i = array_start;  // Restart at the array parent
      } else {
        i              = array_start + 1;  // Restart after the array parent
        array_children = schema[array_start].count;
      }
    }
  }
  return cur;
}

/**
 * @brief Decode column data
 *
 * @param[in] blocks Data block descriptions
 * @param[in] schema Schema description
 * @param[in] global_Dictionary Global dictionary entries
 * @param[in] avro_data Raw block data
 * @param[in] num_blocks Number of blocks
 * @param[in] schema_len Number of entries in schema
 * @param[in] min_row_size Minimum size in bytes of a row
 * @param[in] max_rows Maximum number of rows to load
 * @param[in] first_row Crop all rows below first_row
 *
 **/
// blockDim {32,NWARPS,1}
extern "C" __global__ void __launch_bounds__(NWARPS * 32, 2)
  gpuDecodeAvroColumnData(block_desc_s *blocks,
                          schemadesc_s *schema_g,
                          device_span<nvstrdesc_s> global_dictionary,
                          const uint8_t *avro_data,
                          uint32_t num_blocks,
                          uint32_t schema_len,
                          uint32_t min_row_size,
                          size_t max_rows,
                          size_t first_row)
{
  __shared__ __align__(8) schemadesc_s g_shared_schema[MAX_SHARED_SCHEMA_LEN];
  __shared__ __align__(8) block_desc_s blk_g[NWARPS];

  schemadesc_s *schema;
  block_desc_s *const blk = &blk_g[threadIdx.y];
  uint32_t block_id       = blockIdx.x * NWARPS + threadIdx.y;
  size_t cur_row;
  uint32_t rows_remaining;
  const uint8_t *cur, *end;

  // Fetch schema into shared mem if possible
  if (schema_len <= MAX_SHARED_SCHEMA_LEN) {
    for (int i = threadIdx.y * 32 + threadIdx.x; i < schema_len; i += NWARPS * 32) {
      g_shared_schema[i] = schema_g[i];
    }
    __syncthreads();
    schema = g_shared_schema;
  } else {
    schema = schema_g;
  }
  if (block_id < num_blocks and threadIdx.x == 0) { *blk = blocks[block_id]; }
  __syncthreads();
  if (block_id >= num_blocks) { return; }
  cur_row        = blk->first_row;
  rows_remaining = blk->num_rows;
  cur            = avro_data + blk->offset;
  end            = cur + blk->size;
  while (rows_remaining > 0 && cur < end) {
    uint32_t nrows;
    const uint8_t *start = cur;

    if (cur_row > first_row + max_rows) break;
    if (cur + min_row_size * rows_remaining == end) {
      nrows = min(rows_remaining, 32);
      cur += threadIdx.x * min_row_size;
    } else {
      nrows = 1;
    }
    if (threadIdx.x < nrows) {
      cur = avro_decode_row(schema,
                            schema_g,
                            schema_len,
                            cur_row - first_row + threadIdx.x,
                            max_rows,
                            cur,
                            end,
                            global_dictionary);
    }
    if (nrows <= 1) {
      cur = start + SHFL0(static_cast<uint32_t>(cur - start));
    } else {
      cur = start + nrows * min_row_size;
    }
    SYNCWARP();
    cur_row += nrows;
    rows_remaining -= nrows;
  }
}

/**
 * @brief Launches kernel for decoding column data
 *
 * @param[in] blocks Data block descriptions
 * @param[in] schema Schema description
 * @param[in] global_dictionary Global dictionary entries
 * @param[in] avro_data Raw block data
 * @param[in] num_blocks Number of blocks
 * @param[in] schema_len Number of entries in schema
 * @param[in] max_rows Maximum number of rows to load
 * @param[in] first_row Crop all rows below first_row
 * @param[in] min_row_size Minimum size in bytes of a row
 * @param[in] stream CUDA stream to use, default 0
 */
void __host__ DecodeAvroColumnData(block_desc_s *blocks,
                                   schemadesc_s *schema,
                                   device_span<nvstrdesc_s> global_dictionary,
                                   const uint8_t *avro_data,
                                   uint32_t num_blocks,
                                   uint32_t schema_len,
                                   size_t max_rows,
                                   size_t first_row,
                                   uint32_t min_row_size,
                                   hipStream_t stream)
{
  // NWARPS warps per threadblock
  dim3 const dim_block(32, NWARPS);
  // 1 warp per datablock, NWARPS datablocks per threadblock
  dim3 const dim_grid((num_blocks + NWARPS - 1) / NWARPS, 1);

  gpuDecodeAvroColumnData<<<dim_grid, dim_block, 0, stream>>>(blocks,
                                                              schema,
                                                              global_dictionary,
                                                              avro_data,
                                                              num_blocks,
                                                              schema_len,
                                                              min_row_size,
                                                              max_rows,
                                                              first_row);
}

}  // namespace gpu
}  // namespace avro
}  // namespace io
}  // namespace cudf
