#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <cudf/column/column.hpp>
#include <cudf/column/column_factories.hpp>
#include <cudf/column/column_view.hpp>
#include <cudf/datetime.hpp>
#include <cudf/detail/nvtx/ranges.hpp>
#include <cudf/null_mask.hpp>
#include <cudf/types.hpp>
#include <cudf/utilities/traits.hpp>

#include <rmm/thrust_rmm_allocator.h>

namespace cudf {
namespace datetime {
namespace detail {
template <datetime_component Component>
struct extract_component_operator {
  template <typename Timestamp>
  CUDA_DEVICE_CALLABLE int16_t operator()(Timestamp const ts) const
  {
    using namespace simt::std::chrono;

    auto days_since_epoch = floor<days>(ts);

    auto time_since_midnight = ts - days_since_epoch;

    if (time_since_midnight.count() < 0) { time_since_midnight += days(1); }

    auto hrs_  = duration_cast<hours>(time_since_midnight);
    auto mins_ = duration_cast<minutes>(time_since_midnight - hrs_);
    auto secs_ = duration_cast<seconds>(time_since_midnight - hrs_ - mins_);

    switch (Component) {
      case datetime_component::YEAR:
        return static_cast<int>(year_month_day(days_since_epoch).year());
      case datetime_component::MONTH:
        return static_cast<unsigned>(year_month_day(days_since_epoch).month());
      case datetime_component::DAY:
        return static_cast<unsigned>(year_month_day(days_since_epoch).day());
      case datetime_component::WEEKDAY:
        return year_month_weekday(days_since_epoch).weekday().iso_encoding();
      case datetime_component::HOUR: return hrs_.count();
      case datetime_component::MINUTE: return mins_.count();
      case datetime_component::SECOND: return secs_.count();
      default: return 0;
    }
  }
};

// Round up the date to the last day of the month and return the
// date only (without the time component)
struct extract_last_day_of_month {
  CUDA_DEVICE_CALLABLE auto days_in_month(simt::std::chrono::month mon, bool is_leap_year) const
    -> uint8_t
  {
    using namespace simt::std::chrono;
    // The expression in switch has to be integral/enumerated type.
    // The constexpr in case has to match the switch type
    switch (unsigned{mon}) {
      case unsigned{January}: return 31;
      case unsigned{February}: return is_leap_year ? 29 : 28;
      case unsigned{March}: return 31;
      case unsigned{April}: return 30;
      case unsigned{May}: return 31;
      case unsigned{June}: return 30;
      case unsigned{July}: return 31;
      case unsigned{August}: return 31;
      case unsigned{September}: return 30;
      case unsigned{October}: return 31;
      case unsigned{November}: return 30;
      case unsigned{December}: return 31;
      default: return 0;
    }
  }

  template <typename Timestamp>
  CUDA_DEVICE_CALLABLE timestamp_D operator()(Timestamp const ts) const
  {
    using namespace simt::std::chrono;
    // IDEAL: does not work with CUDA10.0 due to nvcc compiler bug
    // cannot invoke ym_last_day.day()
    // const year_month_day orig_ymd(floor<days>(ts));
    // const year_month_day_last ym_last_day(orig_ymd.year(), month_day_last(orig_ymd.month()));
    // return timestamp_D(sys_days(ym_last_day));

    // Only has the days - time component is chopped off, which is what we want
    auto const days_since_epoch = floor<days>(ts);
    auto const date             = year_month_day(days_since_epoch);
    auto const last_day         = days_in_month(date.month(), date.year().is_leap());

    return timestamp_D(days_since_epoch + days(last_day - static_cast<unsigned>(date.day())));
  }
};

// Number of days until month indexed by leap year and month (0-based index)
static __device__ int16_t const days_until_month[2][12] = {
  {0, 31, 59, 90, 120, 151, 181, 212, 243, 273, 304, 334},  // For non leap years
  {0, 31, 60, 91, 121, 152, 182, 213, 244, 274, 305, 335}   // For leap years
};

// Extract the day number of the year present in the timestamp
struct extract_day_num_of_year {
  template <typename Timestamp>
  CUDA_DEVICE_CALLABLE int16_t operator()(Timestamp const ts) const
  {
    using namespace simt::std::chrono;

    // Only has the days - time component is chopped off, which is what we want
    auto const days_since_epoch = floor<days>(ts);
    auto const date             = year_month_day(days_since_epoch);

    return days_until_month[date.year().is_leap()][unsigned{date.month()} - 1] +
           unsigned{date.day()};
  }
};

// Apply the functor for every element/row in the input column to create the output column
template <typename TransformFunctor, typename OutputColT>
struct launch_functor {
  column_view input;
  mutable_column_view output;

  launch_functor(column_view inp, mutable_column_view out) : input(inp), output(out) {}

  template <typename Element>
  typename std::enable_if_t<!cudf::is_timestamp_t<Element>::value, void> operator()(
    hipStream_t stream) const
  {
    CUDF_FAIL("Cannot extract datetime component from non-timestamp column.");
  }

  template <typename Timestamp>
  typename std::enable_if_t<cudf::is_timestamp_t<Timestamp>::value, void> operator()(
    hipStream_t stream) const
  {
    thrust::transform(rmm::exec_policy(stream)->on(stream),
                      input.begin<Timestamp>(),
                      input.end<Timestamp>(),
                      output.begin<OutputColT>(),
                      TransformFunctor{});
  }
};

// Create an output column by applying the functor to every element from the input column
template <typename TransformFunctor, cudf::type_id OutputColCudfT>
std::unique_ptr<column> apply_datetime_op(column_view const& column,
                                          hipStream_t stream,
                                          rmm::mr::device_memory_resource* mr)
{
  CUDF_EXPECTS(is_timestamp(column.type()), "Column type should be timestamp");
  auto size            = column.size();
  auto output_col_type = data_type{OutputColCudfT};

  // Return an empty column if source column is empty
  if (size == 0) return make_empty_column(output_col_type);

  auto output = make_fixed_width_column(
    output_col_type, size, copy_bitmask(column, stream, mr), column.null_count(), stream, mr);
  auto launch =
    launch_functor<TransformFunctor, typename cudf::id_to_type_impl<OutputColCudfT>::type>{
      column, static_cast<mutable_column_view>(*output)};

  type_dispatcher(column.type(), launch, stream);

  return output;
}

}  // namespace detail

std::unique_ptr<column> extract_year(column_view const& column, rmm::mr::device_memory_resource* mr)
{
  CUDF_FUNC_RANGE();
  return detail::apply_datetime_op<
    detail::extract_component_operator<detail::datetime_component::YEAR>,
    cudf::type_id::INT16>(column, 0, mr);
}

std::unique_ptr<column> extract_month(column_view const& column,
                                      rmm::mr::device_memory_resource* mr)
{
  CUDF_FUNC_RANGE();

  return detail::apply_datetime_op<
    detail::extract_component_operator<detail::datetime_component::MONTH>,
    cudf::type_id::INT16>(column, 0, mr);
}

std::unique_ptr<column> extract_day(column_view const& column, rmm::mr::device_memory_resource* mr)
{
  CUDF_FUNC_RANGE();
  return detail::apply_datetime_op<
    detail::extract_component_operator<detail::datetime_component::DAY>,
    cudf::type_id::INT16>(column, 0, mr);
}

std::unique_ptr<column> extract_weekday(column_view const& column,
                                        rmm::mr::device_memory_resource* mr)
{
  CUDF_FUNC_RANGE();
  return detail::apply_datetime_op<
    detail::extract_component_operator<detail::datetime_component::WEEKDAY>,
    cudf::type_id::INT16>(column, 0, mr);
}

std::unique_ptr<column> extract_hour(column_view const& column, rmm::mr::device_memory_resource* mr)
{
  CUDF_FUNC_RANGE();
  return detail::apply_datetime_op<
    detail::extract_component_operator<detail::datetime_component::HOUR>,
    cudf::type_id::INT16>(column, 0, mr);
}

std::unique_ptr<column> extract_minute(column_view const& column,
                                       rmm::mr::device_memory_resource* mr)
{
  CUDF_FUNC_RANGE();
  return detail::apply_datetime_op<
    detail::extract_component_operator<detail::datetime_component::MINUTE>,
    cudf::type_id::INT16>(column, 0, mr);
}

std::unique_ptr<column> extract_second(column_view const& column,
                                       rmm::mr::device_memory_resource* mr)
{
  CUDF_FUNC_RANGE();
  return detail::apply_datetime_op<
    detail::extract_component_operator<detail::datetime_component::SECOND>,
    cudf::type_id::INT16>(column, 0, mr);
}

std::unique_ptr<column> last_day_of_month(column_view const& column,
                                          rmm::mr::device_memory_resource* mr)
{
  CUDF_FUNC_RANGE();
  return detail::apply_datetime_op<detail::extract_last_day_of_month,
                                   cudf::type_id::TIMESTAMP_DAYS>(column, 0, mr);
}

std::unique_ptr<column> day_of_year(column_view const& column, rmm::mr::device_memory_resource* mr)
{
  CUDF_FUNC_RANGE();
  return detail::apply_datetime_op<detail::extract_day_num_of_year, cudf::type_id::INT16>(
    column, 0, mr);
}

}  // namespace datetime
}  // namespace cudf
