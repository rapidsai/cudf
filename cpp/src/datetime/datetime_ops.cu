#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2020-2022, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <cudf/column/column.hpp>
#include <cudf/column/column_device_view.cuh>
#include <cudf/column/column_factories.hpp>
#include <cudf/column/column_view.hpp>
#include <cudf/datetime.hpp>
#include <cudf/detail/datetime.hpp>
#include <cudf/detail/datetime_ops.cuh>
#include <cudf/detail/indexalator.cuh>
#include <cudf/detail/null_mask.hpp>
#include <cudf/detail/nvtx/ranges.hpp>
#include <cudf/scalar/scalar.hpp>
#include <cudf/table/table_view.hpp>
#include <cudf/types.hpp>
#include <cudf/utilities/default_stream.hpp>
#include <cudf/utilities/error.hpp>
#include <cudf/utilities/traits.hpp>
#include <cudf/utilities/type_dispatcher.hpp>
#include <cudf/wrappers/durations.hpp>

#include <rmm/cuda_stream_view.hpp>
#include <rmm/exec_policy.hpp>

#include <thrust/iterator/constant_iterator.h>
#include <thrust/iterator/permutation_iterator.h>
#include <thrust/transform.h>

namespace cudf {
namespace datetime {
namespace detail {
enum class datetime_component {
  INVALID = 0,
  YEAR,
  MONTH,
  DAY,
  WEEKDAY,
  HOUR,
  MINUTE,
  SECOND,
  MILLISECOND,
  MICROSECOND,
  NANOSECOND
};

enum class rounding_function {
  CEIL,   ///< Rounds up to the next integer multiple of the provided frequency
  FLOOR,  ///< Rounds down to the next integer multiple of the provided frequency
  ROUND   ///< Rounds to the nearest integer multiple of the provided frequency
};

template <datetime_component Component>
struct extract_component_operator {
  template <typename Timestamp>
  __device__ inline int16_t operator()(Timestamp const ts) const
  {
    using namespace cuda::std::chrono;

    auto days_since_epoch = floor<days>(ts);

    auto time_since_midnight = ts - days_since_epoch;

    if (time_since_midnight.count() < 0) { time_since_midnight += days(1); }

    auto const hrs_  = [&] { return duration_cast<hours>(time_since_midnight); };
    auto const mins_ = [&] { return duration_cast<minutes>(time_since_midnight) - hrs_(); };
    auto const secs_ = [&] {
      return duration_cast<seconds>(time_since_midnight) - hrs_() - mins_();
    };
    auto const millisecs_ = [&] {
      return duration_cast<milliseconds>(time_since_midnight) - hrs_() - mins_() - secs_();
    };
    auto const microsecs_ = [&] {
      return duration_cast<microseconds>(time_since_midnight) - hrs_() - mins_() - secs_() -
             millisecs_();
    };
    auto const nanosecs_ = [&] {
      return duration_cast<nanoseconds>(time_since_midnight) - hrs_() - mins_() - secs_() -
             millisecs_() - microsecs_();
    };

    switch (Component) {
      case datetime_component::YEAR:
        return static_cast<int>(year_month_day(days_since_epoch).year());
      case datetime_component::MONTH:
        return static_cast<unsigned>(year_month_day(days_since_epoch).month());
      case datetime_component::DAY:
        return static_cast<unsigned>(year_month_day(days_since_epoch).day());
      case datetime_component::WEEKDAY:
        return year_month_weekday(days_since_epoch).weekday().iso_encoding();
      case datetime_component::HOUR: return hrs_().count();
      case datetime_component::MINUTE: return mins_().count();
      case datetime_component::SECOND: return secs_().count();
      case datetime_component::MILLISECOND: return millisecs_().count();
      case datetime_component::MICROSECOND: return microsecs_().count();
      case datetime_component::NANOSECOND: return nanosecs_().count();
      default: return 0;
    }
  }
};

// This functor takes the rounding type as runtime info and dispatches to the ceil/floor/round
// function.
template <typename DurationType>
struct RoundFunctor {
  template <typename Timestamp>
  __device__ inline auto operator()(rounding_function round_kind, Timestamp dt)
  {
    switch (round_kind) {
      case rounding_function::CEIL: return cuda::std::chrono::ceil<DurationType>(dt);
      case rounding_function::FLOOR: return cuda::std::chrono::floor<DurationType>(dt);
      case rounding_function::ROUND: return cuda::std::chrono::round<DurationType>(dt);
      default: CUDF_UNREACHABLE("Unsupported rounding kind.");
    }
  }
};

struct RoundingDispatcher {
  rounding_function round_kind;
  rounding_frequency component;

  RoundingDispatcher(rounding_function round_kind, rounding_frequency component)
    : round_kind(round_kind), component(component)
  {
  }

  template <typename Timestamp>
  __device__ inline Timestamp operator()(Timestamp const ts) const
  {
    switch (component) {
      case rounding_frequency::DAY:
        return time_point_cast<typename Timestamp::duration>(
          RoundFunctor<duration_D>{}(round_kind, ts));
      case rounding_frequency::HOUR:
        return time_point_cast<typename Timestamp::duration>(
          RoundFunctor<duration_h>{}(round_kind, ts));
      case rounding_frequency::MINUTE:
        return time_point_cast<typename Timestamp::duration>(
          RoundFunctor<duration_m>{}(round_kind, ts));
      case rounding_frequency::SECOND:
        return time_point_cast<typename Timestamp::duration>(
          RoundFunctor<duration_s>{}(round_kind, ts));
      case rounding_frequency::MILLISECOND:
        return time_point_cast<typename Timestamp::duration>(
          RoundFunctor<duration_ms>{}(round_kind, ts));
      case rounding_frequency::MICROSECOND:
        return time_point_cast<typename Timestamp::duration>(
          RoundFunctor<duration_us>{}(round_kind, ts));
      case rounding_frequency::NANOSECOND:
        return time_point_cast<typename Timestamp::duration>(
          RoundFunctor<duration_ns>{}(round_kind, ts));
      default: CUDF_UNREACHABLE("Unsupported datetime rounding resolution.");
    }
  }
};

// Number of days until month indexed by leap year and month (0-based index)
static __device__ int16_t const days_until_month[2][13] = {
  {0, 31, 59, 90, 120, 151, 181, 212, 243, 273, 304, 334, 365},  // For non leap years
  {0, 31, 60, 91, 121, 152, 182, 213, 244, 274, 305, 335, 366}   // For leap years
};

// Round up the date to the last day of the month and return the
// date only (without the time component)
struct extract_last_day_of_month {
  template <typename Timestamp>
  __device__ inline timestamp_D operator()(Timestamp const ts) const
  {
    using namespace cuda::std::chrono;
    const year_month_day ymd(floor<days>(ts));
    auto const ymdl = year_month_day_last{ymd.year() / ymd.month() / last};
    return timestamp_D{sys_days{ymdl}};
  }
};

// Extract the number of days of the month
// A similar operator to `extract_last_day_of_month`, except this returns
// an integer while the other returns a timestamp.
struct days_in_month_op {
  template <typename Timestamp>
  __device__ inline int16_t operator()(Timestamp const ts) const
  {
    using namespace cuda::std::chrono;
    auto const date = year_month_day(floor<days>(ts));
    auto const ymdl = year_month_day_last(date.year() / date.month() / last);
    return static_cast<int16_t>(unsigned{ymdl.day()});
  }
};

// Extract the day number of the year present in the timestamp
struct extract_day_num_of_year {
  template <typename Timestamp>
  __device__ inline int16_t operator()(Timestamp const ts) const
  {
    using namespace cuda::std::chrono;

    // Only has the days - time component is chopped off, which is what we want
    auto const days_since_epoch = floor<days>(ts);
    auto const date             = year_month_day(days_since_epoch);

    return days_until_month[date.year().is_leap()][unsigned{date.month()} - 1] +
           unsigned{date.day()};
  }
};

// Extract the the quarter to which the timestamp belongs to
struct extract_quarter_op {
  template <typename Timestamp>
  __device__ inline int16_t operator()(Timestamp const ts) const
  {
    using namespace cuda::std::chrono;

    // Only has the days - time component is chopped off, which is what we want
    auto const days_since_epoch = floor<days>(ts);
    auto const date             = year_month_day(days_since_epoch);
    auto const month            = unsigned{date.month()};

    // (x + y - 1) / y = ceil(x/y), where x and y are unsigned. x = month, y = 3
    return (month + 2) / 3;
  }
};

// Returns true if the year is a leap year
struct is_leap_year_op {
  template <typename Timestamp>
  __device__ inline bool operator()(Timestamp const ts) const
  {
    using namespace cuda::std::chrono;
    auto const days_since_epoch = floor<days>(ts);
    auto const date             = year_month_day(days_since_epoch);
    return date.year().is_leap();
  }
};

// Specific function for applying ceil/floor/round date ops
struct dispatch_round {
  template <typename Timestamp>
  std::enable_if_t<cudf::is_timestamp<Timestamp>(), std::unique_ptr<cudf::column>> operator()(
    rounding_function round_kind,
    rounding_frequency component,
    cudf::column_view const& column,
    rmm::cuda_stream_view stream,
    rmm::mr::device_memory_resource* mr) const
  {
    auto size            = column.size();
    auto output_col_type = data_type{cudf::type_to_id<Timestamp>()};

    // Return an empty column if source column is empty
    if (size == 0) return make_empty_column(output_col_type);

    auto output = make_fixed_width_column(output_col_type,
                                          size,
                                          cudf::detail::copy_bitmask(column, stream, mr),
                                          column.null_count(),
                                          stream,
                                          mr);

    thrust::transform(rmm::exec_policy(stream),
                      column.begin<Timestamp>(),
                      column.end<Timestamp>(),
                      output->mutable_view().begin<Timestamp>(),
                      RoundingDispatcher{round_kind, component});

    output->set_null_count(column.null_count());

    return output;
  }

  template <typename Timestamp, typename... Args>
  std::enable_if_t<!cudf::is_timestamp<Timestamp>(), std::unique_ptr<cudf::column>> operator()(
    Args&&...)
  {
    CUDF_FAIL("Must be cudf::timestamp");
  }
};

// Apply the functor for every element/row in the input column to create the output column
template <typename TransformFunctor, typename OutputColT>
struct launch_functor {
  column_view input;
  mutable_column_view output;

  launch_functor(column_view inp, mutable_column_view out) : input(inp), output(out) {}

  template <typename Element>
  std::enable_if_t<!cudf::is_timestamp_t<Element>::value, void> operator()(
    rmm::cuda_stream_view stream) const
  {
    CUDF_FAIL("Cannot extract datetime component from non-timestamp column.");
  }

  template <typename Timestamp>
  std::enable_if_t<cudf::is_timestamp_t<Timestamp>::value, void> operator()(
    rmm::cuda_stream_view stream) const
  {
    thrust::transform(rmm::exec_policy(stream),
                      input.begin<Timestamp>(),
                      input.end<Timestamp>(),
                      output.begin<OutputColT>(),
                      TransformFunctor{});
  }
};

// Create an output column by applying the functor to every element from the input column
template <typename TransformFunctor, cudf::type_id OutputColCudfT>
std::unique_ptr<column> apply_datetime_op(column_view const& column,
                                          rmm::cuda_stream_view stream,
                                          rmm::mr::device_memory_resource* mr)
{
  CUDF_EXPECTS(is_timestamp(column.type()), "Column type should be timestamp");
  auto size            = column.size();
  auto output_col_type = data_type{OutputColCudfT};

  // Return an empty column if source column is empty
  if (size == 0) return make_empty_column(output_col_type);

  auto output = make_fixed_width_column(output_col_type,
                                        size,
                                        cudf::detail::copy_bitmask(column, stream, mr),
                                        column.null_count(),
                                        stream,
                                        mr);
  auto launch =
    launch_functor<TransformFunctor, typename cudf::id_to_type_impl<OutputColCudfT>::type>{
      column, static_cast<mutable_column_view>(*output)};

  type_dispatcher(column.type(), launch, stream);

  return output;
}

struct add_calendrical_months_functor {
  template <typename Element, typename... Args>
  std::enable_if_t<!cudf::is_timestamp_t<Element>::value, std::unique_ptr<column>> operator()(
    Args&&...) const
  {
    CUDF_FAIL("Cannot extract datetime component from non-timestamp column.");
  }

  template <typename Timestamp, typename MonthIterator>
  std::enable_if_t<cudf::is_timestamp_t<Timestamp>::value, std::unique_ptr<column>> operator()(
    column_view timestamp_column,
    MonthIterator months_begin,
    rmm::cuda_stream_view stream,
    rmm::mr::device_memory_resource* mr) const
  {
    auto size            = timestamp_column.size();
    auto output_col_type = timestamp_column.type();

    // Return an empty column if source column is empty
    if (size == 0) return make_empty_column(output_col_type);

    // The nullmask of `output` cannot be determined without information from
    // the `months` type (column or scalar). Therefore, it is initialized as
    // `UNALLOCATED` and assigned at a later stage.
    auto output =
      make_fixed_width_column(output_col_type, size, mask_state::UNALLOCATED, stream, mr);
    auto output_mview = output->mutable_view();

    thrust::transform(rmm::exec_policy(stream),
                      timestamp_column.begin<Timestamp>(),
                      timestamp_column.end<Timestamp>(),
                      months_begin,
                      output->mutable_view().begin<Timestamp>(),
                      [] __device__(auto& timestamp, auto& months) {
                        return add_calendrical_months_with_scale_back(
                          timestamp, cuda::std::chrono::months{months});
                      });
    return output;
  }
};

std::unique_ptr<column> add_calendrical_months(column_view const& timestamp_column,
                                               column_view const& months_column,
                                               rmm::cuda_stream_view stream,
                                               rmm::mr::device_memory_resource* mr)
{
  CUDF_EXPECTS(is_timestamp(timestamp_column.type()), "Column type should be timestamp");
  CUDF_EXPECTS(
    months_column.type().id() == type_id::INT16 or months_column.type().id() == type_id::INT32,
    "Months column type should be INT16 or INT32.");
  CUDF_EXPECTS(timestamp_column.size() == months_column.size(),
               "Timestamp and months column should be of the same size");

  auto const months_begin_iter =
    cudf::detail::indexalator_factory::make_input_iterator(months_column);
  auto output = type_dispatcher(timestamp_column.type(),
                                add_calendrical_months_functor{},
                                timestamp_column,
                                months_begin_iter,
                                stream,
                                mr);

  auto [output_null_mask, null_count] =
    cudf::detail::bitmask_and(table_view{{timestamp_column, months_column}}, stream, mr);
  output->set_null_mask(std::move(output_null_mask), null_count);
  return output;
}

std::unique_ptr<column> add_calendrical_months(column_view const& timestamp_column,
                                               scalar const& months,
                                               rmm::cuda_stream_view stream,
                                               rmm::mr::device_memory_resource* mr)
{
  CUDF_EXPECTS(is_timestamp(timestamp_column.type()), "Column type should be timestamp");
  CUDF_EXPECTS(months.type().id() == type_id::INT16 or months.type().id() == type_id::INT32,
               "Months type should be INT16 or INT32");

  if (months.is_valid(stream)) {
    auto const months_begin_iter = thrust::make_permutation_iterator(
      cudf::detail::indexalator_factory::make_input_iterator(months),
      thrust::make_constant_iterator(0));
    auto output = type_dispatcher(timestamp_column.type(),
                                  add_calendrical_months_functor{},
                                  timestamp_column,
                                  months_begin_iter,
                                  stream,
                                  mr);
    output->set_null_mask(cudf::detail::copy_bitmask(timestamp_column, stream, mr));
    return output;
  } else {
    return make_timestamp_column(
      timestamp_column.type(), timestamp_column.size(), mask_state::ALL_NULL, stream, mr);
  }
}

std::unique_ptr<column> round_general(rounding_function round_kind,
                                      rounding_frequency component,
                                      column_view const& column,
                                      rmm::cuda_stream_view stream,
                                      rmm::mr::device_memory_resource* mr)
{
  return cudf::type_dispatcher(
    column.type(), dispatch_round{}, round_kind, component, column, stream, mr);
}

std::unique_ptr<column> extract_year(column_view const& column,
                                     rmm::cuda_stream_view stream,
                                     rmm::mr::device_memory_resource* mr)
{
  return detail::apply_datetime_op<
    detail::extract_component_operator<detail::datetime_component::YEAR>,
    cudf::type_id::INT16>(column, stream, mr);
}

std::unique_ptr<column> extract_month(column_view const& column,
                                      rmm::cuda_stream_view stream,
                                      rmm::mr::device_memory_resource* mr)
{
  return detail::apply_datetime_op<
    detail::extract_component_operator<detail::datetime_component::MONTH>,
    cudf::type_id::INT16>(column, stream, mr);
}

std::unique_ptr<column> extract_day(column_view const& column,
                                    rmm::cuda_stream_view stream,
                                    rmm::mr::device_memory_resource* mr)
{
  return detail::apply_datetime_op<
    detail::extract_component_operator<detail::datetime_component::DAY>,
    cudf::type_id::INT16>(column, stream, mr);
}

std::unique_ptr<column> extract_weekday(column_view const& column,
                                        rmm::cuda_stream_view stream,
                                        rmm::mr::device_memory_resource* mr)
{
  return detail::apply_datetime_op<
    detail::extract_component_operator<detail::datetime_component::WEEKDAY>,
    cudf::type_id::INT16>(column, stream, mr);
}

std::unique_ptr<column> extract_hour(column_view const& column,
                                     rmm::cuda_stream_view stream,
                                     rmm::mr::device_memory_resource* mr)
{
  return detail::apply_datetime_op<
    detail::extract_component_operator<detail::datetime_component::HOUR>,
    cudf::type_id::INT16>(column, stream, mr);
}

std::unique_ptr<column> extract_minute(column_view const& column,
                                       rmm::cuda_stream_view stream,
                                       rmm::mr::device_memory_resource* mr)
{
  return detail::apply_datetime_op<
    detail::extract_component_operator<detail::datetime_component::MINUTE>,
    cudf::type_id::INT16>(column, stream, mr);
}

std::unique_ptr<column> extract_second(column_view const& column,
                                       rmm::cuda_stream_view stream,
                                       rmm::mr::device_memory_resource* mr)
{
  return detail::apply_datetime_op<
    detail::extract_component_operator<detail::datetime_component::SECOND>,
    cudf::type_id::INT16>(column, stream, mr);
}

std::unique_ptr<column> extract_millisecond_fraction(column_view const& column,
                                                     rmm::cuda_stream_view stream,
                                                     rmm::mr::device_memory_resource* mr)
{
  return detail::apply_datetime_op<
    detail::extract_component_operator<detail::datetime_component::MILLISECOND>,
    cudf::type_id::INT16>(column, stream, mr);
}

std::unique_ptr<column> extract_microsecond_fraction(column_view const& column,
                                                     rmm::cuda_stream_view stream,
                                                     rmm::mr::device_memory_resource* mr)
{
  return detail::apply_datetime_op<
    detail::extract_component_operator<detail::datetime_component::MICROSECOND>,
    cudf::type_id::INT16>(column, stream, mr);
}

std::unique_ptr<column> extract_nanosecond_fraction(column_view const& column,
                                                    rmm::cuda_stream_view stream,
                                                    rmm::mr::device_memory_resource* mr)
{
  return detail::apply_datetime_op<
    detail::extract_component_operator<detail::datetime_component::NANOSECOND>,
    cudf::type_id::INT16>(column, stream, mr);
}

std::unique_ptr<column> last_day_of_month(column_view const& column,
                                          rmm::cuda_stream_view stream,
                                          rmm::mr::device_memory_resource* mr)
{
  return detail::apply_datetime_op<detail::extract_last_day_of_month,
                                   cudf::type_id::TIMESTAMP_DAYS>(column, stream, mr);
}

std::unique_ptr<column> day_of_year(column_view const& column,
                                    rmm::cuda_stream_view stream,
                                    rmm::mr::device_memory_resource* mr)
{
  return detail::apply_datetime_op<detail::extract_day_num_of_year, cudf::type_id::INT16>(
    column, stream, mr);
}

std::unique_ptr<column> is_leap_year(column_view const& column,
                                     rmm::cuda_stream_view stream,
                                     rmm::mr::device_memory_resource* mr)
{
  return apply_datetime_op<is_leap_year_op, type_id::BOOL8>(column, stream, mr);
}

std::unique_ptr<column> days_in_month(column_view const& column,
                                      rmm::cuda_stream_view stream,
                                      rmm::mr::device_memory_resource* mr)
{
  return apply_datetime_op<days_in_month_op, type_id::INT16>(column, stream, mr);
}

std::unique_ptr<column> extract_quarter(column_view const& column,
                                        rmm::cuda_stream_view stream,
                                        rmm::mr::device_memory_resource* mr)
{
  return apply_datetime_op<extract_quarter_op, type_id::INT16>(column, stream, mr);
}

}  // namespace detail

std::unique_ptr<column> ceil_datetimes(column_view const& column,
                                       rounding_frequency freq,
                                       rmm::mr::device_memory_resource* mr)
{
  CUDF_FUNC_RANGE();
  return detail::round_general(
    detail::rounding_function::CEIL, freq, column, cudf::get_default_stream(), mr);
}

std::unique_ptr<column> floor_datetimes(column_view const& column,
                                        rounding_frequency freq,
                                        rmm::mr::device_memory_resource* mr)
{
  CUDF_FUNC_RANGE();
  return detail::round_general(
    detail::rounding_function::FLOOR, freq, column, cudf::get_default_stream(), mr);
}

std::unique_ptr<column> round_datetimes(column_view const& column,
                                        rounding_frequency freq,
                                        rmm::mr::device_memory_resource* mr)
{
  CUDF_FUNC_RANGE();
  return detail::round_general(
    detail::rounding_function::ROUND, freq, column, cudf::get_default_stream(), mr);
}

std::unique_ptr<column> extract_year(column_view const& column, rmm::mr::device_memory_resource* mr)
{
  CUDF_FUNC_RANGE();
  return detail::extract_year(column, cudf::get_default_stream(), mr);
}

std::unique_ptr<column> extract_month(column_view const& column,
                                      rmm::mr::device_memory_resource* mr)
{
  CUDF_FUNC_RANGE();
  return detail::extract_month(column, cudf::get_default_stream(), mr);
}

std::unique_ptr<column> extract_day(column_view const& column, rmm::mr::device_memory_resource* mr)
{
  CUDF_FUNC_RANGE();
  return detail::extract_day(column, cudf::get_default_stream(), mr);
}

std::unique_ptr<column> extract_weekday(column_view const& column,
                                        rmm::mr::device_memory_resource* mr)
{
  CUDF_FUNC_RANGE();
  return detail::extract_weekday(column, cudf::get_default_stream(), mr);
}

std::unique_ptr<column> extract_hour(column_view const& column, rmm::mr::device_memory_resource* mr)
{
  CUDF_FUNC_RANGE();
  return detail::extract_hour(column, cudf::get_default_stream(), mr);
}

std::unique_ptr<column> extract_minute(column_view const& column,
                                       rmm::mr::device_memory_resource* mr)
{
  CUDF_FUNC_RANGE();
  return detail::extract_minute(column, cudf::get_default_stream(), mr);
}

std::unique_ptr<column> extract_second(column_view const& column,
                                       rmm::mr::device_memory_resource* mr)
{
  CUDF_FUNC_RANGE();
  return detail::extract_second(column, cudf::get_default_stream(), mr);
}

std::unique_ptr<column> extract_millisecond_fraction(column_view const& column,
                                                     rmm::mr::device_memory_resource* mr)
{
  CUDF_FUNC_RANGE();
  return detail::extract_millisecond_fraction(column, cudf::get_default_stream(), mr);
}

std::unique_ptr<column> extract_microsecond_fraction(column_view const& column,
                                                     rmm::mr::device_memory_resource* mr)
{
  CUDF_FUNC_RANGE();
  return detail::extract_microsecond_fraction(column, cudf::get_default_stream(), mr);
}

std::unique_ptr<column> extract_nanosecond_fraction(column_view const& column,
                                                    rmm::mr::device_memory_resource* mr)
{
  CUDF_FUNC_RANGE();
  return detail::extract_nanosecond_fraction(column, cudf::get_default_stream(), mr);
}

std::unique_ptr<column> last_day_of_month(column_view const& column,
                                          rmm::mr::device_memory_resource* mr)
{
  CUDF_FUNC_RANGE();
  return detail::last_day_of_month(column, cudf::get_default_stream(), mr);
}

std::unique_ptr<column> day_of_year(column_view const& column, rmm::mr::device_memory_resource* mr)
{
  CUDF_FUNC_RANGE();
  return detail::day_of_year(column, cudf::get_default_stream(), mr);
}

std::unique_ptr<cudf::column> add_calendrical_months(cudf::column_view const& timestamp_column,
                                                     cudf::column_view const& months_column,
                                                     rmm::mr::device_memory_resource* mr)
{
  CUDF_FUNC_RANGE();
  return detail::add_calendrical_months(
    timestamp_column, months_column, cudf::get_default_stream(), mr);
}

std::unique_ptr<cudf::column> add_calendrical_months(cudf::column_view const& timestamp_column,
                                                     cudf::scalar const& months,
                                                     rmm::mr::device_memory_resource* mr)
{
  CUDF_FUNC_RANGE();
  return detail::add_calendrical_months(timestamp_column, months, cudf::get_default_stream(), mr);
}

std::unique_ptr<column> is_leap_year(column_view const& column, rmm::mr::device_memory_resource* mr)
{
  CUDF_FUNC_RANGE();
  return detail::is_leap_year(column, cudf::get_default_stream(), mr);
}

std::unique_ptr<column> days_in_month(column_view const& column,
                                      rmm::mr::device_memory_resource* mr)
{
  CUDF_FUNC_RANGE();
  return detail::days_in_month(column, cudf::get_default_stream(), mr);
}

std::unique_ptr<column> extract_quarter(column_view const& column,
                                        rmm::mr::device_memory_resource* mr)
{
  CUDF_FUNC_RANGE();
  return detail::extract_quarter(column, cudf::get_default_stream(), mr);
}

}  // namespace datetime
}  // namespace cudf
