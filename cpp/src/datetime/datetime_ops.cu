#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2020, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <cudf/column/column.hpp>
#include <cudf/column/column_device_view.cuh>
#include <cudf/column/column_factories.hpp>
#include <cudf/column/column_view.hpp>
#include <cudf/datetime.hpp>
#include <cudf/detail/null_mask.hpp>
#include <cudf/detail/nvtx/ranges.hpp>
#include <cudf/null_mask.hpp>
#include <cudf/table/table_view.hpp>
#include <cudf/types.hpp>
#include <cudf/utilities/traits.hpp>

#include <rmm/thrust_rmm_allocator.h>
#include <rmm/cuda_stream_view.hpp>

namespace cudf {
namespace datetime {
namespace detail {
enum class datetime_component {
  INVALID = 0,
  YEAR,
  MONTH,
  DAY,
  WEEKDAY,
  HOUR,
  MINUTE,
  SECOND,
};

template <datetime_component Component>
struct extract_component_operator {
  template <typename Timestamp>
  CUDA_DEVICE_CALLABLE int16_t operator()(Timestamp const ts) const
  {
    using namespace simt::std::chrono;

    auto days_since_epoch = floor<days>(ts);

    auto time_since_midnight = ts - days_since_epoch;

    if (time_since_midnight.count() < 0) { time_since_midnight += days(1); }

    auto hrs_  = duration_cast<hours>(time_since_midnight);
    auto mins_ = duration_cast<minutes>(time_since_midnight - hrs_);
    auto secs_ = duration_cast<seconds>(time_since_midnight - hrs_ - mins_);

    switch (Component) {
      case datetime_component::YEAR:
        return static_cast<int>(year_month_day(days_since_epoch).year());
      case datetime_component::MONTH:
        return static_cast<unsigned>(year_month_day(days_since_epoch).month());
      case datetime_component::DAY:
        return static_cast<unsigned>(year_month_day(days_since_epoch).day());
      case datetime_component::WEEKDAY:
        return year_month_weekday(days_since_epoch).weekday().iso_encoding();
      case datetime_component::HOUR: return hrs_.count();
      case datetime_component::MINUTE: return mins_.count();
      case datetime_component::SECOND: return secs_.count();
      default: return 0;
    }
  }
};

// Number of days until month indexed by leap year and month (0-based index)
static __device__ int16_t const days_until_month[2][13] = {
  {0, 31, 59, 90, 120, 151, 181, 212, 243, 273, 304, 334, 365},  // For non leap years
  {0, 31, 60, 91, 121, 152, 182, 213, 244, 274, 305, 335, 366}   // For leap years
};

CUDA_DEVICE_CALLABLE uint8_t days_in_month(simt::std::chrono::month mon, bool is_leap_year)
{
  return days_until_month[is_leap_year][unsigned{mon}] -
         days_until_month[is_leap_year][unsigned{mon} - 1];
}

// Round up the date to the last day of the month and return the
// date only (without the time component)
struct extract_last_day_of_month {
  template <typename Timestamp>
  CUDA_DEVICE_CALLABLE timestamp_D operator()(Timestamp const ts) const
  {
    using namespace simt::std::chrono;
    // IDEAL: does not work with CUDA10.0 due to nvcc compiler bug
    // cannot invoke ym_last_day.day()
    // const year_month_day orig_ymd(floor<days>(ts));
    // const year_month_day_last ym_last_day(orig_ymd.year(), month_day_last(orig_ymd.month()));
    // return timestamp_D(sys_days(ym_last_day));

    // Only has the days - time component is chopped off, which is what we want
    auto const days_since_epoch = floor<days>(ts);
    auto const date             = year_month_day(days_since_epoch);
    auto const last_day         = days_in_month(date.month(), date.year().is_leap());

    return timestamp_D(days_since_epoch + days(last_day - static_cast<unsigned>(date.day())));
  }
};

// Extract the day number of the year present in the timestamp
struct extract_day_num_of_year {
  template <typename Timestamp>
  CUDA_DEVICE_CALLABLE int16_t operator()(Timestamp const ts) const
  {
    using namespace simt::std::chrono;

    // Only has the days - time component is chopped off, which is what we want
    auto const days_since_epoch = floor<days>(ts);
    auto const date             = year_month_day(days_since_epoch);

    return days_until_month[date.year().is_leap()][unsigned{date.month()} - 1] +
           unsigned{date.day()};
  }
};

// Apply the functor for every element/row in the input column to create the output column
template <typename TransformFunctor, typename OutputColT>
struct launch_functor {
  column_view input;
  mutable_column_view output;

  launch_functor(column_view inp, mutable_column_view out) : input(inp), output(out) {}

  template <typename Element>
  typename std::enable_if_t<!cudf::is_timestamp_t<Element>::value, void> operator()(
    hipStream_t stream) const
  {
    CUDF_FAIL("Cannot extract datetime component from non-timestamp column.");
  }

  template <typename Timestamp>
  typename std::enable_if_t<cudf::is_timestamp_t<Timestamp>::value, void> operator()(
    hipStream_t stream) const
  {
    thrust::transform(rmm::exec_policy(stream)->on(stream),
                      input.begin<Timestamp>(),
                      input.end<Timestamp>(),
                      output.begin<OutputColT>(),
                      TransformFunctor{});
  }
};

// Create an output column by applying the functor to every element from the input column
template <typename TransformFunctor, cudf::type_id OutputColCudfT>
std::unique_ptr<column> apply_datetime_op(column_view const& column,
                                          hipStream_t stream,
                                          rmm::mr::device_memory_resource* mr)
{
  CUDF_EXPECTS(is_timestamp(column.type()), "Column type should be timestamp");
  auto size            = column.size();
  auto output_col_type = data_type{OutputColCudfT};

  // Return an empty column if source column is empty
  if (size == 0) return make_empty_column(output_col_type);

  auto output =
    make_fixed_width_column(output_col_type,
                            size,
                            cudf::detail::copy_bitmask(column, rmm::cuda_stream_view{stream}, mr),
                            column.null_count(),
                            stream,
                            mr);
  auto launch =
    launch_functor<TransformFunctor, typename cudf::id_to_type_impl<OutputColCudfT>::type>{
      column, static_cast<mutable_column_view>(*output)};

  type_dispatcher(column.type(), launch, stream);

  return output;
}

struct add_calendrical_months_functor {
  column_view timestamp_column;
  column_view months_column;
  mutable_column_view output;

  add_calendrical_months_functor(column_view tsc, column_view mc, mutable_column_view out)
    : timestamp_column(tsc), months_column(mc), output(out)
  {
  }

  // std chrono implementation is copied here due to nvcc bug 2909685
  // https://howardhinnant.github.io/date_algorithms.html#days_from_civil
  static CUDA_DEVICE_CALLABLE timestamp_D
  compute_sys_days(simt::std::chrono::year_month_day const& ymd)
  {
    const int yr = static_cast<int>(ymd.year()) - (ymd.month() <= simt::std::chrono::month{2});
    const unsigned mth = static_cast<unsigned>(ymd.month());
    const unsigned dy  = static_cast<unsigned>(ymd.day());

    const int era      = (yr >= 0 ? yr : yr - 399) / 400;
    const unsigned yoe = static_cast<unsigned>(yr - era * 400);                // [0, 399]
    const unsigned doy = (153 * (mth + (mth > 2 ? -3 : 9)) + 2) / 5 + dy - 1;  // [0, 365]
    const unsigned doe = yoe * 365 + yoe / 4 - yoe / 100 + doy;                // [0, 146096]
    return timestamp_D{duration_D{era * 146097 + static_cast<int>(doe) - 719468}};
  }

  template <typename Element>
  typename std::enable_if_t<!cudf::is_timestamp_t<Element>::value, void> operator()(
    hipStream_t stream) const
  {
    CUDF_FAIL("Cannot extract datetime component from non-timestamp column.");
  }

  template <typename Timestamp>
  typename std::enable_if_t<cudf::is_timestamp_t<Timestamp>::value, void> operator()(
    hipStream_t stream) const
  {
    thrust::transform(rmm::exec_policy(stream)->on(stream),
                      timestamp_column.begin<Timestamp>(),
                      timestamp_column.end<Timestamp>(),
                      months_column.begin<int16_t>(),
                      output.begin<Timestamp>(),
                      [] __device__(auto time_val, auto months_val) {
                        using namespace simt::std::chrono;
                        using duration_m = duration<int32_t, months::period>;

                        // Get the days component from the input
                        auto days_since_epoch = floor<days>(time_val);

                        // Add the number of months
                        year_month_day ymd{days_since_epoch};
                        ymd += duration_m{months_val};

                        // If the new date isn't valid, scale it back to the last day of the
                        // month.
                        // IDEAL: if (!ymd.ok()) ymd = ymd.year()/ymd.month()/last;
                        auto month_days = days_in_month(ymd.month(), ymd.year().is_leap());
                        if (unsigned{ymd.day()} > month_days)
                          ymd = ymd.year() / ymd.month() / day{month_days};

                        // Put back the time component to the date
                        return
                          // IDEAL: sys_days{ymd} + ...
                          compute_sys_days(ymd) + (time_val - days_since_epoch);
                      });
  }
};

std::unique_ptr<column> add_calendrical_months(column_view const& timestamp_column,
                                               column_view const& months_column,
                                               hipStream_t stream,
                                               rmm::mr::device_memory_resource* mr)
{
  CUDF_EXPECTS(is_timestamp(timestamp_column.type()), "Column type should be timestamp");
  CUDF_EXPECTS(months_column.type() == data_type{type_id::INT16},
               "Months column type should be INT16");
  CUDF_EXPECTS(timestamp_column.size() == months_column.size(),
               "Timestamp and months column should be of the same size");
  auto size            = timestamp_column.size();
  auto output_col_type = timestamp_column.type();

  // Return an empty column if source column is empty
  if (size == 0) return make_empty_column(output_col_type);

  auto output_col_mask =
    cudf::detail::bitmask_and(table_view({timestamp_column, months_column}), stream, mr);
  auto output = make_fixed_width_column(
    output_col_type, size, std::move(output_col_mask), cudf::UNKNOWN_NULL_COUNT, stream, mr);

  auto launch = add_calendrical_months_functor{
    timestamp_column, months_column, static_cast<mutable_column_view>(*output)};

  type_dispatcher(timestamp_column.type(), launch, stream);

  return output;
}
}  // namespace detail

std::unique_ptr<column> extract_year(column_view const& column, rmm::mr::device_memory_resource* mr)
{
  CUDF_FUNC_RANGE();
  return detail::apply_datetime_op<
    detail::extract_component_operator<detail::datetime_component::YEAR>,
    cudf::type_id::INT16>(column, 0, mr);
}

std::unique_ptr<column> extract_month(column_view const& column,
                                      rmm::mr::device_memory_resource* mr)
{
  CUDF_FUNC_RANGE();

  return detail::apply_datetime_op<
    detail::extract_component_operator<detail::datetime_component::MONTH>,
    cudf::type_id::INT16>(column, 0, mr);
}

std::unique_ptr<column> extract_day(column_view const& column, rmm::mr::device_memory_resource* mr)
{
  CUDF_FUNC_RANGE();
  return detail::apply_datetime_op<
    detail::extract_component_operator<detail::datetime_component::DAY>,
    cudf::type_id::INT16>(column, 0, mr);
}

std::unique_ptr<column> extract_weekday(column_view const& column,
                                        rmm::mr::device_memory_resource* mr)
{
  CUDF_FUNC_RANGE();
  return detail::apply_datetime_op<
    detail::extract_component_operator<detail::datetime_component::WEEKDAY>,
    cudf::type_id::INT16>(column, 0, mr);
}

std::unique_ptr<column> extract_hour(column_view const& column, rmm::mr::device_memory_resource* mr)
{
  CUDF_FUNC_RANGE();
  return detail::apply_datetime_op<
    detail::extract_component_operator<detail::datetime_component::HOUR>,
    cudf::type_id::INT16>(column, 0, mr);
}

std::unique_ptr<column> extract_minute(column_view const& column,
                                       rmm::mr::device_memory_resource* mr)
{
  CUDF_FUNC_RANGE();
  return detail::apply_datetime_op<
    detail::extract_component_operator<detail::datetime_component::MINUTE>,
    cudf::type_id::INT16>(column, 0, mr);
}

std::unique_ptr<column> extract_second(column_view const& column,
                                       rmm::mr::device_memory_resource* mr)
{
  CUDF_FUNC_RANGE();
  return detail::apply_datetime_op<
    detail::extract_component_operator<detail::datetime_component::SECOND>,
    cudf::type_id::INT16>(column, 0, mr);
}

std::unique_ptr<column> last_day_of_month(column_view const& column,
                                          rmm::mr::device_memory_resource* mr)
{
  CUDF_FUNC_RANGE();
  return detail::apply_datetime_op<detail::extract_last_day_of_month,
                                   cudf::type_id::TIMESTAMP_DAYS>(column, 0, mr);
}

std::unique_ptr<column> day_of_year(column_view const& column, rmm::mr::device_memory_resource* mr)
{
  CUDF_FUNC_RANGE();
  return detail::apply_datetime_op<detail::extract_day_num_of_year, cudf::type_id::INT16>(
    column, 0, mr);
}

std::unique_ptr<cudf::column> add_calendrical_months(cudf::column_view const& timestamp_column,
                                                     cudf::column_view const& months_column,
                                                     rmm::mr::device_memory_resource* mr)
{
  CUDF_FUNC_RANGE();
  return detail::add_calendrical_months(timestamp_column, months_column, 0, mr);
}
}  // namespace datetime
}  // namespace cudf
