#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2020-2021, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <cudf/column/column.hpp>
#include <cudf/column/column_device_view.cuh>
#include <cudf/column/column_factories.hpp>
#include <cudf/column/column_view.hpp>
#include <cudf/datetime.hpp>
#include <cudf/detail/datetime.hpp>
#include <cudf/detail/null_mask.hpp>
#include <cudf/detail/nvtx/ranges.hpp>
#include <cudf/table/table_view.hpp>
#include <cudf/types.hpp>
#include <cudf/utilities/error.hpp>
#include <cudf/utilities/traits.hpp>
#include <cudf/utilities/type_dispatcher.hpp>
#include <cudf/wrappers/durations.hpp>

#include <rmm/cuda_stream_view.hpp>
#include <rmm/exec_policy.hpp>

namespace cudf {
namespace datetime {
namespace detail {
enum class datetime_component {
  INVALID = 0,
  YEAR,
  MONTH,
  DAY,
  WEEKDAY,
  HOUR,
  MINUTE,
  SECOND,
  MILLISECOND,
  MICROSECOND,
  NANOSECOND
};

template <datetime_component Component>
struct extract_component_operator {
  template <typename Timestamp>
  CUDA_DEVICE_CALLABLE int16_t operator()(Timestamp const ts) const
  {
    using namespace cuda::std::chrono;

    auto days_since_epoch = floor<days>(ts);

    auto time_since_midnight = ts - days_since_epoch;

    if (time_since_midnight.count() < 0) { time_since_midnight += days(1); }

    auto hrs_  = duration_cast<hours>(time_since_midnight);
    auto mins_ = duration_cast<minutes>(time_since_midnight - hrs_);
    auto secs_ = duration_cast<seconds>(time_since_midnight - hrs_ - mins_);

    switch (Component) {
      case datetime_component::YEAR:
        return static_cast<int>(year_month_day(days_since_epoch).year());
      case datetime_component::MONTH:
        return static_cast<unsigned>(year_month_day(days_since_epoch).month());
      case datetime_component::DAY:
        return static_cast<unsigned>(year_month_day(days_since_epoch).day());
      case datetime_component::WEEKDAY:
        return year_month_weekday(days_since_epoch).weekday().iso_encoding();
      case datetime_component::HOUR: return hrs_.count();
      case datetime_component::MINUTE: return mins_.count();
      case datetime_component::SECOND: return secs_.count();
      default: return 0;
    }
  }
};

template <datetime_component COMPONENT>
struct ceil_timestamp {
  template <typename Timestamp>
  CUDA_DEVICE_CALLABLE Timestamp operator()(Timestamp const ts) const
  {
    using namespace cuda::std::chrono;
    // want to use this with D, H, T (minute), S, L (millisecond), U
    switch (COMPONENT) {
      case datetime_component::DAY:
        return time_point_cast<typename Timestamp::duration>(ceil<duration_D>(ts));
      case datetime_component::HOUR:
        return time_point_cast<typename Timestamp::duration>(ceil<duration_h>(ts));
      case datetime_component::MINUTE:
        return time_point_cast<typename Timestamp::duration>(ceil<duration_m>(ts));
      case datetime_component::SECOND:
        return time_point_cast<typename Timestamp::duration>(ceil<duration_s>(ts));
      case datetime_component::MILLISECOND:
        return time_point_cast<typename Timestamp::duration>(ceil<duration_ms>(ts));
      case datetime_component::MICROSECOND:
        return time_point_cast<typename Timestamp::duration>(ceil<duration_us>(ts));
      case datetime_component::NANOSECOND:
        return time_point_cast<typename Timestamp::duration>(ceil<duration_ns>(ts));
      default: cudf_assert(false && "Unexpected resolution");
    }

    return {};
  }
};

// Number of days until month indexed by leap year and month (0-based index)
static __device__ int16_t const days_until_month[2][13] = {
  {0, 31, 59, 90, 120, 151, 181, 212, 243, 273, 304, 334, 365},  // For non leap years
  {0, 31, 60, 91, 121, 152, 182, 213, 244, 274, 305, 335, 366}   // For leap years
};

// Round up the date to the last day of the month and return the
// date only (without the time component)
struct extract_last_day_of_month {
  template <typename Timestamp>
  CUDA_DEVICE_CALLABLE timestamp_D operator()(Timestamp const ts) const
  {
    using namespace cuda::std::chrono;
    const year_month_day ymd(floor<days>(ts));
    auto const ymdl = year_month_day_last{ymd.year() / ymd.month() / last};
    return timestamp_D{sys_days{ymdl}};
  }
};

// Extract the number of days of the month
// A similar operator to `extract_last_day_of_month`, except this returns
// an integer while the other returns a timestamp.
struct days_in_month_op {
  template <typename Timestamp>
  CUDA_DEVICE_CALLABLE int16_t operator()(Timestamp const ts) const
  {
    using namespace cuda::std::chrono;
    auto const date = year_month_day(floor<days>(ts));
    auto const ymdl = year_month_day_last(date.year() / date.month() / last);
    return static_cast<int16_t>(unsigned{ymdl.day()});
  }
};

// Extract the day number of the year present in the timestamp
struct extract_day_num_of_year {
  template <typename Timestamp>
  CUDA_DEVICE_CALLABLE int16_t operator()(Timestamp const ts) const
  {
    using namespace cuda::std::chrono;

    // Only has the days - time component is chopped off, which is what we want
    auto const days_since_epoch = floor<days>(ts);
    auto const date             = year_month_day(days_since_epoch);

    return days_until_month[date.year().is_leap()][unsigned{date.month()} - 1] +
           unsigned{date.day()};
  }
};

// Extract the the quarter to which the timestamp belongs to
struct extract_quarter_op {
  template <typename Timestamp>
  CUDA_DEVICE_CALLABLE int16_t operator()(Timestamp const ts) const
  {
    using namespace cuda::std::chrono;

    // Only has the days - time component is chopped off, which is what we want
    auto const days_since_epoch = floor<days>(ts);
    auto const date             = year_month_day(days_since_epoch);
    auto const month            = unsigned{date.month()};

    // (x + y - 1) / y = ceil(x/y), where x and y are unsigned. x = month, y = 3
    return (month + 2) / 3;
  }
};

// Returns true if the year is a leap year
struct is_leap_year_op {
  template <typename Timestamp>
  CUDA_DEVICE_CALLABLE bool operator()(Timestamp const ts) const
  {
    using namespace cuda::std::chrono;
    auto const days_since_epoch = floor<days>(ts);
    auto const date             = year_month_day(days_since_epoch);
    return date.year().is_leap();
  }
};

// Specific function for applying ceil/floor date ops
template <typename TransformFunctor>
struct dispatch_ceil {
  template <typename Timestamp>
  std::enable_if_t<cudf::is_timestamp<Timestamp>(), std::unique_ptr<cudf::column>> operator()(
    cudf::column_view const& column,
    rmm::cuda_stream_view stream,
    rmm::mr::device_memory_resource* mr) const
  {
    auto size            = column.size();
    auto output_col_type = data_type{cudf::type_to_id<Timestamp>()};

    // Return an empty column if source column is empty
    if (size == 0) return make_empty_column(output_col_type);

    auto output = make_fixed_width_column(output_col_type,
                                          size,
                                          cudf::detail::copy_bitmask(column, stream, mr),
                                          column.null_count(),
                                          stream,
                                          mr);

    thrust::transform(rmm::exec_policy(stream),
                      column.begin<Timestamp>(),
                      column.end<Timestamp>(),
                      output->mutable_view().begin<Timestamp>(),
                      TransformFunctor{});

    return output;
  }

  template <typename Timestamp, typename... Args>
  std::enable_if_t<!cudf::is_timestamp<Timestamp>(), std::unique_ptr<cudf::column>> operator()(
    Args&&...)
  {
    CUDF_FAIL("Must be cudf::timestamp");
  }
};

// Apply the functor for every element/row in the input column to create the output column
template <typename TransformFunctor, typename OutputColT>
struct launch_functor {
  column_view input;
  mutable_column_view output;

  launch_functor(column_view inp, mutable_column_view out) : input(inp), output(out) {}

  template <typename Element>
  typename std::enable_if_t<!cudf::is_timestamp_t<Element>::value, void> operator()(
    rmm::cuda_stream_view stream) const
  {
    CUDF_FAIL("Cannot extract datetime component from non-timestamp column.");
  }

  template <typename Timestamp>
  typename std::enable_if_t<cudf::is_timestamp_t<Timestamp>::value, void> operator()(
    rmm::cuda_stream_view stream) const
  {
    thrust::transform(rmm::exec_policy(stream),
                      input.begin<Timestamp>(),
                      input.end<Timestamp>(),
                      output.begin<OutputColT>(),
                      TransformFunctor{});
  }
};

// Create an output column by applying the functor to every element from the input column
template <typename TransformFunctor, cudf::type_id OutputColCudfT>
std::unique_ptr<column> apply_datetime_op(column_view const& column,
                                          rmm::cuda_stream_view stream,
                                          rmm::mr::device_memory_resource* mr)
{
  CUDF_EXPECTS(is_timestamp(column.type()), "Column type should be timestamp");
  auto size            = column.size();
  auto output_col_type = data_type{OutputColCudfT};

  // Return an empty column if source column is empty
  if (size == 0) return make_empty_column(output_col_type);

  auto output = make_fixed_width_column(output_col_type,
                                        size,
                                        cudf::detail::copy_bitmask(column, stream, mr),
                                        column.null_count(),
                                        stream,
                                        mr);
  auto launch =
    launch_functor<TransformFunctor, typename cudf::id_to_type_impl<OutputColCudfT>::type>{
      column, static_cast<mutable_column_view>(*output)};

  type_dispatcher(column.type(), launch, stream);

  return output;
}

struct add_calendrical_months_functor {
  column_view timestamp_column;
  column_view months_column;
  mutable_column_view output;

  add_calendrical_months_functor(column_view tsc, column_view mc, mutable_column_view out)
    : timestamp_column(tsc), months_column(mc), output(out)
  {
  }

  template <typename Element>
  typename std::enable_if_t<!cudf::is_timestamp_t<Element>::value, void> operator()(
    rmm::cuda_stream_view stream) const
  {
    CUDF_FAIL("Cannot extract datetime component from non-timestamp column.");
  }

  template <typename Timestamp>
  typename std::enable_if_t<cudf::is_timestamp_t<Timestamp>::value, void> operator()(
    rmm::cuda_stream_view stream) const
  {
    thrust::transform(rmm::exec_policy(stream),
                      timestamp_column.begin<Timestamp>(),
                      timestamp_column.end<Timestamp>(),
                      months_column.begin<int16_t>(),
                      output.begin<Timestamp>(),
                      [] __device__(auto time_val, auto months_val) {
                        using namespace cuda::std::chrono;
                        using duration_m = duration<int32_t, months::period>;

                        // Get the days component from the input
                        auto days_since_epoch = floor<days>(time_val);

                        // Add the number of months
                        year_month_day ymd{days_since_epoch};
                        ymd += duration_m{months_val};

                        // If the new date isn't valid, scale it back to the last day of the
                        // month.
                        if (!ymd.ok()) ymd = ymd.year() / ymd.month() / last;

                        // Put back the time component to the date
                        return sys_days{ymd} + (time_val - days_since_epoch);
                      });
  }
};

std::unique_ptr<column> add_calendrical_months(column_view const& timestamp_column,
                                               column_view const& months_column,
                                               rmm::cuda_stream_view stream,
                                               rmm::mr::device_memory_resource* mr)
{
  CUDF_EXPECTS(is_timestamp(timestamp_column.type()), "Column type should be timestamp");
  CUDF_EXPECTS(months_column.type() == data_type{type_id::INT16},
               "Months column type should be INT16");
  CUDF_EXPECTS(timestamp_column.size() == months_column.size(),
               "Timestamp and months column should be of the same size");
  auto size            = timestamp_column.size();
  auto output_col_type = timestamp_column.type();

  // Return an empty column if source column is empty
  if (size == 0) return make_empty_column(output_col_type);

  auto output_col_mask =
    cudf::detail::bitmask_and(table_view({timestamp_column, months_column}), stream, mr);
  auto output = make_fixed_width_column(
    output_col_type, size, std::move(output_col_mask), cudf::UNKNOWN_NULL_COUNT, stream, mr);

  auto launch = add_calendrical_months_functor{
    timestamp_column, months_column, static_cast<mutable_column_view>(*output)};

  type_dispatcher(timestamp_column.type(), launch, stream);

  return output;
}

template <datetime_component Component>
std::unique_ptr<column> ceil_general(column_view const& column,
                                     rmm::cuda_stream_view stream,
                                     rmm::mr::device_memory_resource* mr)
{
  return cudf::type_dispatcher(
    column.type(), dispatch_ceil<detail::ceil_timestamp<Component>>{}, column, stream, mr);
}

std::unique_ptr<column> extract_year(column_view const& column,
                                     rmm::cuda_stream_view stream,
                                     rmm::mr::device_memory_resource* mr)
{
  return detail::apply_datetime_op<
    detail::extract_component_operator<detail::datetime_component::YEAR>,
    cudf::type_id::INT16>(column, stream, mr);
}

std::unique_ptr<column> extract_month(column_view const& column,
                                      rmm::cuda_stream_view stream,
                                      rmm::mr::device_memory_resource* mr)
{
  return detail::apply_datetime_op<
    detail::extract_component_operator<detail::datetime_component::MONTH>,
    cudf::type_id::INT16>(column, stream, mr);
}

std::unique_ptr<column> extract_day(column_view const& column,
                                    rmm::cuda_stream_view stream,
                                    rmm::mr::device_memory_resource* mr)
{
  return detail::apply_datetime_op<
    detail::extract_component_operator<detail::datetime_component::DAY>,
    cudf::type_id::INT16>(column, stream, mr);
}

std::unique_ptr<column> extract_weekday(column_view const& column,
                                        rmm::cuda_stream_view stream,
                                        rmm::mr::device_memory_resource* mr)
{
  return detail::apply_datetime_op<
    detail::extract_component_operator<detail::datetime_component::WEEKDAY>,
    cudf::type_id::INT16>(column, stream, mr);
}

std::unique_ptr<column> extract_hour(column_view const& column,
                                     rmm::cuda_stream_view stream,
                                     rmm::mr::device_memory_resource* mr)
{
  return detail::apply_datetime_op<
    detail::extract_component_operator<detail::datetime_component::HOUR>,
    cudf::type_id::INT16>(column, stream, mr);
}

std::unique_ptr<column> extract_minute(column_view const& column,
                                       rmm::cuda_stream_view stream,
                                       rmm::mr::device_memory_resource* mr)
{
  return detail::apply_datetime_op<
    detail::extract_component_operator<detail::datetime_component::MINUTE>,
    cudf::type_id::INT16>(column, stream, mr);
}

std::unique_ptr<column> extract_second(column_view const& column,
                                       rmm::cuda_stream_view stream,
                                       rmm::mr::device_memory_resource* mr)
{
  return detail::apply_datetime_op<
    detail::extract_component_operator<detail::datetime_component::SECOND>,
    cudf::type_id::INT16>(column, stream, mr);
}

std::unique_ptr<column> last_day_of_month(column_view const& column,
                                          rmm::cuda_stream_view stream,
                                          rmm::mr::device_memory_resource* mr)
{
  return detail::apply_datetime_op<detail::extract_last_day_of_month,
                                   cudf::type_id::TIMESTAMP_DAYS>(column, stream, mr);
}

std::unique_ptr<column> day_of_year(column_view const& column,
                                    rmm::cuda_stream_view stream,
                                    rmm::mr::device_memory_resource* mr)
{
  return detail::apply_datetime_op<detail::extract_day_num_of_year, cudf::type_id::INT16>(
    column, stream, mr);
}

std::unique_ptr<column> is_leap_year(column_view const& column,
                                     rmm::cuda_stream_view stream,
                                     rmm::mr::device_memory_resource* mr)
{
  return apply_datetime_op<is_leap_year_op, type_id::BOOL8>(column, stream, mr);
}

std::unique_ptr<column> days_in_month(column_view const& column,
                                      rmm::cuda_stream_view stream,
                                      rmm::mr::device_memory_resource* mr)
{
  return apply_datetime_op<days_in_month_op, type_id::INT16>(column, stream, mr);
}

std::unique_ptr<column> extract_quarter(column_view const& column,
                                        rmm::cuda_stream_view stream,
                                        rmm::mr::device_memory_resource* mr)
{
  return apply_datetime_op<extract_quarter_op, type_id::INT16>(column, stream, mr);
}

}  // namespace detail

std::unique_ptr<column> ceil_day(column_view const& column, rmm::mr::device_memory_resource* mr)
{
  CUDF_FUNC_RANGE();
  return detail::ceil_general<detail::datetime_component::DAY>(
    column, rmm::cuda_stream_default, mr);
}

std::unique_ptr<column> ceil_hour(column_view const& column, rmm::mr::device_memory_resource* mr)
{
  CUDF_FUNC_RANGE();
  return detail::ceil_general<detail::datetime_component::HOUR>(
    column, rmm::cuda_stream_default, mr);
}

std::unique_ptr<column> ceil_minute(column_view const& column, rmm::mr::device_memory_resource* mr)
{
  CUDF_FUNC_RANGE();
  return detail::ceil_general<detail::datetime_component::MINUTE>(
    column, rmm::cuda_stream_default, mr);
}

std::unique_ptr<column> ceil_second(column_view const& column, rmm::mr::device_memory_resource* mr)
{
  CUDF_FUNC_RANGE();
  return detail::ceil_general<detail::datetime_component::SECOND>(
    column, rmm::cuda_stream_default, mr);
}

std::unique_ptr<column> ceil_millisecond(column_view const& column,
                                         rmm::mr::device_memory_resource* mr)
{
  CUDF_FUNC_RANGE();
  return detail::ceil_general<detail::datetime_component::MILLISECOND>(
    column, rmm::cuda_stream_default, mr);
}

std::unique_ptr<column> ceil_microsecond(column_view const& column,
                                         rmm::mr::device_memory_resource* mr)
{
  CUDF_FUNC_RANGE();
  return detail::ceil_general<detail::datetime_component::MICROSECOND>(
    column, rmm::cuda_stream_default, mr);
}

std::unique_ptr<column> ceil_nanosecond(column_view const& column,
                                        rmm::mr::device_memory_resource* mr)
{
  CUDF_FUNC_RANGE();
  return detail::ceil_general<detail::datetime_component::NANOSECOND>(
    column, rmm::cuda_stream_default, mr);
}

std::unique_ptr<column> extract_year(column_view const& column, rmm::mr::device_memory_resource* mr)
{
  CUDF_FUNC_RANGE();
  return detail::extract_year(column, rmm::cuda_stream_default, mr);
}

std::unique_ptr<column> extract_month(column_view const& column,
                                      rmm::mr::device_memory_resource* mr)
{
  CUDF_FUNC_RANGE();
  return detail::extract_month(column, rmm::cuda_stream_default, mr);
}

std::unique_ptr<column> extract_day(column_view const& column, rmm::mr::device_memory_resource* mr)
{
  CUDF_FUNC_RANGE();
  return detail::extract_day(column, rmm::cuda_stream_default, mr);
}

std::unique_ptr<column> extract_weekday(column_view const& column,
                                        rmm::mr::device_memory_resource* mr)
{
  CUDF_FUNC_RANGE();
  return detail::extract_weekday(column, rmm::cuda_stream_default, mr);
}

std::unique_ptr<column> extract_hour(column_view const& column, rmm::mr::device_memory_resource* mr)
{
  CUDF_FUNC_RANGE();
  return detail::extract_hour(column, rmm::cuda_stream_default, mr);
}

std::unique_ptr<column> extract_minute(column_view const& column,
                                       rmm::mr::device_memory_resource* mr)
{
  CUDF_FUNC_RANGE();
  return detail::extract_minute(column, rmm::cuda_stream_default, mr);
}

std::unique_ptr<column> extract_second(column_view const& column,
                                       rmm::mr::device_memory_resource* mr)
{
  CUDF_FUNC_RANGE();
  return detail::extract_second(column, rmm::cuda_stream_default, mr);
}

std::unique_ptr<column> last_day_of_month(column_view const& column,
                                          rmm::mr::device_memory_resource* mr)
{
  CUDF_FUNC_RANGE();
  return detail::last_day_of_month(column, rmm::cuda_stream_default, mr);
}

std::unique_ptr<column> day_of_year(column_view const& column, rmm::mr::device_memory_resource* mr)
{
  CUDF_FUNC_RANGE();
  return detail::day_of_year(column, rmm::cuda_stream_default, mr);
}

std::unique_ptr<cudf::column> add_calendrical_months(cudf::column_view const& timestamp_column,
                                                     cudf::column_view const& months_column,
                                                     rmm::mr::device_memory_resource* mr)
{
  CUDF_FUNC_RANGE();
  return detail::add_calendrical_months(
    timestamp_column, months_column, rmm::cuda_stream_default, mr);
}

std::unique_ptr<column> is_leap_year(column_view const& column, rmm::mr::device_memory_resource* mr)
{
  CUDF_FUNC_RANGE();
  return detail::is_leap_year(column, rmm::cuda_stream_default, mr);
}

std::unique_ptr<column> days_in_month(column_view const& column,
                                      rmm::mr::device_memory_resource* mr)
{
  CUDF_FUNC_RANGE();
  return detail::days_in_month(column, rmm::cuda_stream_default, mr);
}

std::unique_ptr<column> extract_quarter(column_view const& column,
                                        rmm::mr::device_memory_resource* mr)
{
  CUDF_FUNC_RANGE();
  return detail::extract_quarter(column, rmm::cuda_stream_default, mr);
}

}  // namespace datetime
}  // namespace cudf
