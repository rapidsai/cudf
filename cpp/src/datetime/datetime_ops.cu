#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <cudf/types.hpp>
#include <cudf/datetime.hpp>
#include <cudf/null_mask.hpp>
#include <cudf/utilities/traits.hpp>
#include <cudf/column/column.hpp>
#include <cudf/column/column_view.hpp>
#include <cudf/detail/nvtx/ranges.hpp>

#include <rmm/thrust_rmm_allocator.h>

namespace cudf {
namespace datetime {
namespace detail {

template <typename Timestamp, datetime_component Component>
struct extract_component_operator {
  static_assert(cudf::is_timestamp<Timestamp>(), "");

  CUDA_DEVICE_CALLABLE int16_t operator()(Timestamp const ts) const {
    using namespace simt::std::chrono;

    auto days_since_epoch = floor<days>(ts);

    auto time_since_midnight = ts - days_since_epoch;

    if (time_since_midnight.count() < 0) {
      time_since_midnight += days(1);
    }

    auto hrs_ = duration_cast<hours>(time_since_midnight);
    auto mins_ = duration_cast<minutes>(time_since_midnight - hrs_);
    auto secs_ = duration_cast<seconds>(time_since_midnight - hrs_ - mins_);

    switch (Component) {
      case datetime_component::YEAR:
        return static_cast<int>(year_month_day(days_since_epoch).year());
      case datetime_component::MONTH:
        return static_cast<unsigned>(year_month_day(days_since_epoch).month());
      case datetime_component::DAY:
        return static_cast<unsigned>(year_month_day(days_since_epoch).day());
      case datetime_component::WEEKDAY:
        return year_month_weekday(days_since_epoch).weekday().iso_encoding();
      case datetime_component::HOUR:
        return hrs_.count();
      case datetime_component::MINUTE:
        return mins_.count();
      case datetime_component::SECOND:
        return secs_.count();
      default:
        return 0;
    }
  }
};

template <datetime_component Component>
struct launch_extract_component {
  column_view input;
  mutable_column_view output;

  launch_extract_component(column_view inp, mutable_column_view out)
      : input(inp), output(out) {}

  template <typename Element>
  typename std::enable_if_t<!cudf::is_timestamp_t<Element>::value, void>
  operator()(hipStream_t stream) {
    CUDF_FAIL("Cannot extract datetime component from non-timestamp column.");
  }

  template <typename Timestamp>
  typename std::enable_if_t<cudf::is_timestamp_t<Timestamp>::value, void>
  operator()(hipStream_t stream) {
    thrust::transform(rmm::exec_policy(stream)->on(stream),
                      input.begin<Timestamp>(), input.end<Timestamp>(),
                      output.begin<int16_t>(),
                      extract_component_operator<Timestamp, Component>{});
  }
};

template <datetime_component Component>
std::unique_ptr<column> extract_component(column_view const& column,
                                          hipStream_t stream,
                                          rmm::mr::device_memory_resource* mr) {
  auto size = column.size();
  auto type = data_type{type_id::INT16};
  auto null_mask = copy_bitmask(column, stream, mr);
  auto output = std::make_unique<cudf::column>(
      type, size, rmm::device_buffer{size * cudf::size_of(type), stream, mr},
      null_mask, column.null_count(),
      std::vector<std::unique_ptr<cudf::column>>{});

  auto launch = launch_extract_component<Component>{
      column, static_cast<mutable_column_view>(*output)};

  experimental::type_dispatcher(column.type(), launch, stream);

  return output;
}
}  // namespace detail

std::unique_ptr<column> extract_year(column_view const& column,
                                     rmm::mr::device_memory_resource* mr) {
  CUDF_FUNC_RANGE();
  return detail::extract_component<detail::datetime_component::YEAR>(column, 0,
                                                                     mr);
}

std::unique_ptr<column> extract_month(column_view const& column,
                                      rmm::mr::device_memory_resource* mr) {
  CUDF_FUNC_RANGE();
  return detail::extract_component<detail::datetime_component::MONTH>(column, 0,
                                                                      mr);
}

std::unique_ptr<column> extract_day(column_view const& column,
                                    rmm::mr::device_memory_resource* mr) {
  CUDF_FUNC_RANGE();
  return detail::extract_component<detail::datetime_component::DAY>(column, 0,
                                                                    mr);
}

std::unique_ptr<column> extract_weekday(column_view const& column,
                                        rmm::mr::device_memory_resource* mr) {
  CUDF_FUNC_RANGE();
  return detail::extract_component<detail::datetime_component::WEEKDAY>(column,
                                                                        0, mr);
}

std::unique_ptr<column> extract_hour(column_view const& column,
                                     rmm::mr::device_memory_resource* mr) {
  CUDF_FUNC_RANGE();
  return detail::extract_component<detail::datetime_component::HOUR>(column, 0,
                                                                     mr);
}

std::unique_ptr<column> extract_minute(column_view const& column,
                                       rmm::mr::device_memory_resource* mr) {
  CUDF_FUNC_RANGE();
  return detail::extract_component<detail::datetime_component::MINUTE>(column,
                                                                       0, mr);
}

std::unique_ptr<column> extract_second(column_view const& column,
                                       rmm::mr::device_memory_resource* mr) {
  CUDF_FUNC_RANGE();
  return detail::extract_component<detail::datetime_component::SECOND>(column,
                                                                       0, mr);
}

}  // namespace datetime
}  // namespace cudf
