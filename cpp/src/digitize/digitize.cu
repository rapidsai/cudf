/*
 * Copyright (c) 2019, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */
#include "digitize.hpp"
#include <cudf/digitize.hpp>
#include <cudf/column/column_factories.hpp>
#include <cudf/utilities/traits.hpp>
#include <cudf/utilities/type_dispatcher.hpp>
#include <utilities/error_utils.hpp>
#include <thrust/binary_search.h>

namespace cudf {

namespace {

struct binary_search_bound {
  template<typename T>
  auto operator()(column_view const& col, column_view const& bins, bool upper_bound,
                  rmm::mr::device_memory_resource* mr, hipStream_t stream)
  {
    auto output = cudf::make_numeric_column(data_type{INT32}, col.size(), cudf::UNALLOCATED, stream, mr);

    if (upper_bound) {
      thrust::upper_bound(rmm::exec_policy()->on(stream), bins.begin<T>(), bins.end<T>(),
        col.begin<T>(), col.end<T>(), output.begin<int32_t>(), thrust::less_equal<T>());
    } else {
      thrust::lower_bound(rmm::exec_policy()->on(stream), bins.begin<T>(), bins.end<T>(),
        col.begin<T>(), col.end<T>(), output.begin<int32_t>(), thrust::less_equal<T>());
    }

    return output;
  }
};

}  // namespace

namespace detail {

std::unique_ptr<column>
digitize(column_view const& col, column_view const& bins, bool right,
         rmm::mr::device_memory_resource* mr, hipStream_t stream)
{
  auto const dtype = col.type();
  CUDF_EXPECTS(dtype == bins.type(), "Column type mismatch");

  // TODO should this make use of the comparable type traits?
  CUDF_EXPECTS(is_numeric(dtype) || is_timestamp(dtype), "Type must be numeric or timestamp");

  // TODO: Handle when col or bins have null values
  CUDF_EXPECTS(0 == col.null_count(), "Null values unsupported");
  CUDF_EXPECTS(0 == bins.null_count(), "Null values unsupported");

  return experimental::type_dispatcher(dtype, binary_search_bound{},
    col, bins, right, stream);
}

}  // namespace detail

std::unique_ptr<column>
digitize(column_view const& col, column_view const& bins, bool right,
         rmm::mr::device_memory_resource* mr)
{
  return detail::digitize(col, bins, right, mr);
}

}  // namespace cudf
