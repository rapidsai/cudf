#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2018, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */
/** ---------------------------------------------------------------------------*
 * @brief Operations on GDF column validity bitmasks
 * 
 * @file column.cpp
 * ---------------------------------------------------------------------------**/
#include <vector>
#include <cassert>
#include <hipcub/hipcub.hpp>

#include "cudf.h"
#include "rmm/rmm.h"
#include "rmm/thrust_rmm_allocator.h"
#include "utilities/error_utils.h"
#include "utilities/cudf_utils.h"

#include <thrust/tabulate.h>


// To account for if gdf_valid_type is not a 4 byte type,
// compute the RATIO of the number of bytes in gdf_valid_type
// to the 4 byte type being used for casting
using valid32_t = uint32_t;
constexpr size_t RATIO = sizeof(valid32_t) / sizeof(gdf_valid_type);
constexpr int BITS_PER_MASK32 = GDF_VALID_BITSIZE * RATIO;

constexpr int block_size = 256;

/** --------------------------------------------------------------------------*
 * @Synopsis  Counts the number of valid bits for the specified number of rows
 * in the host vector of gdf_valid_type masks
 * 
 * @Param masks The host vector of masks whose bits will be counted
 * @Param num_rows The number of bits to count
 * 
 * @Returns  The number of valid bits in [0, num_rows) in the host vector of masks
 * ----------------------------------------------------------------------------*/
size_t count_valid_bits_host(std::vector<gdf_valid_type> const & masks, int const num_rows)
{
  if((0 == num_rows) || (0 == masks.size())){
    return 0;
  }

  size_t count{0};

  // Count the valid bits for all masks except the last one
  for(size_t i = 0; i < (masks.size() - 1); ++i)
  {
    gdf_valid_type current_mask = masks[i];

    while(current_mask > 0)
    {
      current_mask &= (current_mask-1) ;
      count++;
    }
  }

  // Only count the bits in the last mask that correspond to rows
  int num_rows_last_mask = num_rows % GDF_VALID_BITSIZE;

  if(num_rows_last_mask == 0)
    num_rows_last_mask = GDF_VALID_BITSIZE;

  gdf_valid_type last_mask = *(masks.end() - 1);
  for(int i = 0; (i < num_rows_last_mask) && (last_mask > 0); ++i)
  {
    count += (last_mask & gdf_valid_type(1));
    last_mask >>= 1;
  }

  return count;
}


/* --------------------------------------------------------------------------*/
/** 
 * @brief Kernel to count the number of set bits in a column's validity buffer
 *
 * The underlying buffer type may only be a 1B type, but it is casted to a 4B 
 * type (valid32_t) such that __popc may be used to more efficiently count the 
 * number of set bits. This requires handling the last 4B element as a special 
 * case as the buffer may not be a multiple of 4 bytes.
 * 
 * @Param[in] masks32 Pointer to buffer (casted as a 4B type) whose bits will be counted
 * @Param[in] num_masks32 The number of 4B elements in the buffer
 * @Param[in] num_rows The number of rows in the column, i.e., the number of bits
 * in the buffer that correspond to rows
 * @Param[out] global_count The number of set bits in the range of bits [0, num_rows)
 */
/* ----------------------------------------------------------------------------*/
template <typename size_type>
__global__ 
void count_valid_bits(valid32_t const * const masks32, 
                      int const num_masks32, 
                      int const num_rows, 
                      size_type * const global_count)
{
  using BlockReduce = hipcub::BlockReduce<size_type, block_size>;
  __shared__ typename BlockReduce::TempStorage temp_storage;

  // If the number of rows is not a multiple of 32, then the remaining 
  // rows need to be handled separtely because not all of its bits correspond
  // to rows
  int last_mask32{0};
  int const num_rows_last_mask{num_rows % BITS_PER_MASK32};
  if(0 == num_rows_last_mask)
    last_mask32 = num_masks32;
  else
    last_mask32 = num_masks32 - 1;

  int const idx{static_cast<int>(threadIdx.x + blockIdx.x * blockDim.x)};

  int cur_mask{idx};

  size_type my_count{0};

  // Use popc to count the valid bits for the all of the masks 
  // where all of the bits correspond to rows
  while(cur_mask < last_mask32)
  {
    my_count += __popc(masks32[cur_mask]);
    cur_mask += blockDim.x * gridDim.x;
  }

  // Handle the remainder rows
  if(idx < num_rows_last_mask)
  {
    gdf_valid_type const * const valids{reinterpret_cast<gdf_valid_type const *>(masks32)};
    int const my_row{num_rows - idx - 1};

    if(true == gdf_is_valid(valids,my_row))
      ++my_count;
  }

  // Reduces the count from each thread in a block into a block count
  int const block_count{BlockReduce(temp_storage).Sum(my_count)};

  // Store the block count into the global count
  if(threadIdx.x == 0)
  {
    atomicAdd(global_count, block_count);
  }
}

/* ---------------------------------------------------------------------------*
 * @Synopsis  Counts the number of valid bits for the specified number of rows
 * in a validity bitmask.
 * 
 * @Param[in] masks The validity bitmask buffer in device memory
 * @Param[in] num_rows The number of bits to count
 * @Param[out] count The number of valid bits in the buffer from [0, num_rows)
 * 
 * @Returns  GDF_SUCCESS upon successful completion 
 *
 * ----------------------------------------------------------------------------*/
gdf_error gdf_count_nonzero_mask(gdf_valid_type const *masks,
                                 gdf_size_type num_rows, gdf_size_type *count) {
  if((nullptr == masks) || (nullptr == count)){return GDF_DATASET_EMPTY;}
  if(0 == num_rows) {return GDF_SUCCESS;}

  // Masks will be proccessed as 4B types, therefore we require that the underlying
  // type be less than or equal to 4B
  assert(sizeof(valid32_t) >= sizeof(gdf_valid_type));

  // Number of gdf_valid_types in the validity bitmask
  gdf_size_type const num_masks{gdf_get_num_chars_bitmask(num_rows)};

  // Number of 4 byte types in the validity bit mask 
  gdf_size_type num_masks32{static_cast<gdf_size_type>(std::ceil(static_cast<float>(num_masks) / RATIO))};

  int h_count{0};
  if(num_masks32 > 0)
  {
    // TODO: Probably shouldn't create/destroy the stream every time
    hipStream_t count_stream;
    CUDA_TRY(hipStreamCreate(&count_stream));
    int * d_count{nullptr};

    // Cast validity buffer to 4 byte type
    valid32_t const * masks32{reinterpret_cast<valid32_t const *>(masks)};

    RMM_TRY(RMM_ALLOC((void**)&d_count, sizeof(int), count_stream));
    CUDA_TRY(hipMemsetAsync(d_count, 0, sizeof(int), count_stream));

    gdf_size_type const grid_size{(num_masks32 + block_size - 1)/block_size};

    count_valid_bits<<<grid_size, block_size,0,count_stream>>>(masks32, num_masks32, num_rows, d_count);

    CUDA_TRY( hipGetLastError() );

    CUDA_TRY(hipMemcpyAsync(&h_count, d_count, sizeof(int), hipMemcpyDeviceToHost, count_stream));
    RMM_TRY(RMM_FREE(d_count, count_stream));
    CUDA_TRY(hipStreamSynchronize(count_stream));
    CUDA_TRY(hipStreamDestroy(count_stream));
  }

  assert(h_count >= 0);
  assert(h_count <= num_rows);

  *count = h_count;

  return GDF_SUCCESS;
}

/** ---------------------------------------------------------------------------*
 * @brief Concatenate the validity bitmasks of multiple columns
 * 
 * Accounts for the differences between lengths of columns and their bitmasks 
 * (e.g. because gdf_valid_type is larger than one bit).
 * 
 * @param[out] output_mask The concatenated mask
 * @param[in] output_column_length The total length (in data elements) of the 
 *                                 concatenated column
 * @param[in] masks_to_concat The array of device pointers to validity bitmasks
 *                            for the columns to concatenate
 * @param[in] column_lengths An array of lengths of the columns to concatenate
 * @param[in] num_columns The number of columns to concatenate
 * @return gdf_error GDF_SUCCESS or GDF_CUDA_ERROR if there is a runtime CUDA
           error
 * ---------------------------------------------------------------------------**/
gdf_error gdf_mask_concat(gdf_valid_type *output_mask,
                          gdf_size_type output_column_length,            
                          gdf_valid_type *masks_to_concat[], 
                          gdf_size_type *column_lengths, 
                          gdf_size_type num_columns)
{
    // This lambda is executed in a thrust algorithm. Each thread computes and
    // returns one gdf_valid_type element for the concatenated output mask
    auto mask_concatenator = [=] __device__ (gdf_size_type mask_index) {
      gdf_valid_type output_m = 0;
     
      int cur_mask_index = 0, cur_mask_start = 0;
      int cur_mask_len = column_lengths[0];
      
      // Each thread processes one GDF_VALID_BITSIZE worth of valid bits
      for (int bit = 0; bit < GDF_VALID_BITSIZE; ++bit) 
      { 
        gdf_size_type output_index = mask_index * GDF_VALID_BITSIZE + bit;

        // stop when we are beyond the length of the output column (in elements)
        if (output_index >= output_column_length) break;
        
        // find the next column's mask when we step past the current column's length
        while ( (cur_mask_start + cur_mask_len <= output_index) && (cur_mask_index < num_columns - 1) )
        {
          cur_mask_start += cur_mask_len;
          cur_mask_len = column_lengths[++cur_mask_index];           
        }
        
        // Set each valid bit at the right location in this thread's output gdf_valid_type
        // Note: gdf_is_valid returns true when the input mask is a null pointer
        // This makes it behave as if columns with null validity masks have masks of all 1s,
        // which is the desired behavior.
        gdf_size_type index = output_index - cur_mask_start;
        if ( gdf_is_valid(masks_to_concat[cur_mask_index], index) ) 
        {
          output_m |= (1 << bit);     
        }
      }

      return output_m;
    };

    // This is like thrust::for_each where the lambda gets the current index into the output array
    // as input
    thrust::tabulate(rmm::exec_policy()->on(0),
                     output_mask,
                     output_mask + gdf_get_num_chars_bitmask(output_column_length),
                     mask_concatenator);

    CUDA_TRY( hipGetLastError() );
        
    return GDF_SUCCESS;
}


