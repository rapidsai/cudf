#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019-2020, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <cudf/column/column_device_view.cuh>
#include <cudf/detail/null_mask.hpp>
#include <cudf/detail/nvtx/ranges.hpp>
#include <cudf/detail/utilities/cuda.cuh>
#include <cudf/detail/utilities/integer_utils.hpp>
#include <cudf/null_mask.hpp>
#include <cudf/table/table_view.hpp>
#include <cudf/utilities/bit.hpp>
#include <cudf/utilities/error.hpp>

#include <rmm/cuda_stream_view.hpp>
#include <rmm/device_buffer.hpp>
#include <rmm/device_scalar.hpp>
#include <rmm/device_vector.hpp>
#include <rmm/mr/device/device_memory_resource.hpp>

#include <thrust/binary_search.h>
#include <thrust/copy.h>
#include <thrust/device_ptr.h>
#include <thrust/extrema.h>

#include <hipcub/hipcub.hpp>

#include <algorithm>
#include <numeric>
#include <type_traits>

namespace cudf {
size_type state_null_count(mask_state state, size_type size)
{
  switch (state) {
    case mask_state::UNALLOCATED: return 0;
    case mask_state::UNINITIALIZED: return UNKNOWN_NULL_COUNT;
    case mask_state::ALL_NULL: return size;
    case mask_state::ALL_VALID: return 0;
    default: CUDF_FAIL("Invalid null mask state.");
  }
}

// Computes required allocation size of a bitmask
std::size_t bitmask_allocation_size_bytes(size_type number_of_bits, std::size_t padding_boundary)
{
  CUDF_EXPECTS(padding_boundary > 0, "Invalid padding boundary");
  auto necessary_bytes = cudf::util::div_rounding_up_safe<size_type>(number_of_bits, CHAR_BIT);

  auto padded_bytes = padding_boundary * cudf::util::div_rounding_up_safe<size_type>(
                                           necessary_bytes, padding_boundary);
  return padded_bytes;
}

// Computes number of *actual* bitmask_type elements needed
size_type num_bitmask_words(size_type number_of_bits)
{
  return cudf::util::div_rounding_up_safe<size_type>(number_of_bits,
                                                     detail::size_in_bits<bitmask_type>());
}

namespace detail {

// Create a device_buffer for a null mask
rmm::device_buffer create_null_mask(size_type size,
                                    mask_state state,
                                    rmm::cuda_stream_view stream,
                                    rmm::mr::device_memory_resource *mr)
{
  size_type mask_size{0};

  if (state != mask_state::UNALLOCATED) { mask_size = bitmask_allocation_size_bytes(size); }

  rmm::device_buffer mask(mask_size, stream, mr);

  if (state != mask_state::UNINITIALIZED) {
    uint8_t fill_value = (state == mask_state::ALL_VALID) ? 0xff : 0x00;
    CUDA_TRY(hipMemsetAsync(
      static_cast<bitmask_type *>(mask.data()), fill_value, mask_size, stream.value()));
  }

  return mask;
}

namespace {
__global__ void set_null_mask_kernel(bitmask_type *__restrict__ destination,
                                     size_type begin_bit,
                                     size_type end_bit,
                                     bool valid,
                                     size_type number_of_mask_words)
{
  auto x                  = destination + word_index(begin_bit);
  const auto last_word    = word_index(end_bit) - word_index(begin_bit);
  bitmask_type fill_value = (valid == true) ? 0xffffffff : 0x00;

  for (size_type destination_word_index = threadIdx.x + blockIdx.x * blockDim.x;
       destination_word_index < number_of_mask_words;
       destination_word_index += blockDim.x * gridDim.x) {
    if (destination_word_index == 0 || destination_word_index == last_word) {
      bitmask_type mask = ~bitmask_type{0};
      if (destination_word_index == 0) {
        mask = ~(set_least_significant_bits(intra_word_index(begin_bit)));
      }
      if (destination_word_index == last_word) {
        mask = mask & set_least_significant_bits(intra_word_index(end_bit));
      }
      x[destination_word_index] =
        (valid == true) ? x[destination_word_index] | mask : x[destination_word_index] & ~mask;
    } else {
      x[destination_word_index] = fill_value;
    }
  }
}
}  // namespace

// Set pre-allocated null mask of given bit range [begin_bit, end_bit) to valid, if valid==true,
// or null, otherwise;
void set_null_mask(bitmask_type *bitmask,
                   size_type begin_bit,
                   size_type end_bit,
                   bool valid,
                   rmm::cuda_stream_view stream)
{
  CUDF_FUNC_RANGE();
  CUDF_EXPECTS(begin_bit >= 0, "Invalid range.");
  CUDF_EXPECTS(begin_bit < end_bit, "Invalid bit range.");
  if (bitmask != nullptr) {
    auto number_of_mask_words =
      num_bitmask_words(end_bit) - begin_bit / detail::size_in_bits<bitmask_type>();
    cudf::detail::grid_1d config(number_of_mask_words, 256);
    set_null_mask_kernel<<<config.num_blocks, config.num_threads_per_block, 0, stream.value()>>>(
      static_cast<bitmask_type *>(bitmask), begin_bit, end_bit, valid, number_of_mask_words);
    CHECK_CUDA(stream.value());
  }
}

}  // namespace detail

// Create a device_buffer for a null mask
rmm::device_buffer create_null_mask(size_type size,
                                    mask_state state,
                                    rmm::mr::device_memory_resource *mr)
{
  return detail::create_null_mask(size, state, rmm::cuda_stream_default, mr);
}

// Set pre-allocated null mask of given bit range [begin_bit, end_bit) to valid, if valid==true,
// or null, otherwise;
void set_null_mask(bitmask_type *bitmask, size_type begin_bit, size_type end_bit, bool valid)
{
  return detail::set_null_mask(bitmask, begin_bit, end_bit, valid);
}

namespace {

/**
 * @brief Counts the number of non-zero bits in a bitmask in the range
 * `[first_bit_index, last_bit_index]`.
 *
 * Expects `0 <= first_bit_index <= last_bit_index`.
 *
 * @param[in] bitmask The bitmask whose non-zero bits will be counted.
 * @param[in] first_bit_index The index (inclusive) of the first bit to count
 * @param[in] last_bit_index The index (inclusive) of the last bit to count
 * @param[out] global_count The number of non-zero bits in the specified range
 */
template <size_type block_size>
__global__ void count_set_bits_kernel(bitmask_type const *bitmask,
                                      size_type first_bit_index,
                                      size_type last_bit_index,
                                      size_type *global_count)
{
  constexpr auto const word_size{detail::size_in_bits<bitmask_type>()};

  auto const first_word_index{word_index(first_bit_index)};
  auto const last_word_index{word_index(last_bit_index)};
  auto const tid         = threadIdx.x + blockIdx.x * blockDim.x;
  auto thread_word_index = tid + first_word_index;
  size_type thread_count{0};

  // First, just count the bits in all words
  while (thread_word_index <= last_word_index) {
    thread_count += __popc(bitmask[thread_word_index]);
    thread_word_index += blockDim.x * gridDim.x;
  }

  // Subtract any slack bits counted from the first and last word
  // Two threads handle this -- one for first word, one for last
  if (tid < 2) {
    bool const first{tid == 0};
    bool const last{not first};

    size_type bit_index  = (first) ? first_bit_index : last_bit_index;
    size_type word_index = (first) ? first_word_index : last_word_index;

    size_type num_slack_bits = bit_index % word_size;
    if (last) { num_slack_bits = word_size - num_slack_bits - 1; }

    if (num_slack_bits > 0) {
      bitmask_type word = bitmask[word_index];
      auto slack_mask   = (first) ? set_least_significant_bits(num_slack_bits)
                                : set_most_significant_bits(num_slack_bits);

      thread_count -= __popc(word & slack_mask);
    }
  }

  using BlockReduce = hipcub::BlockReduce<size_type, block_size>;
  __shared__ typename BlockReduce::TempStorage temp_storage;
  size_type block_count{BlockReduce(temp_storage).Sum(thread_count)};

  if (threadIdx.x == 0) { atomicAdd(global_count, block_count); }
}

/**
 * For each range `[first_bit_indices[i], last_bit_indices[i])`
 * (where 0 <= i < `num_ranges`), count the number of bits set outside the range
 * in the boundary words (i.e. words that include either
 * `first_bit_indices[i]'th` bit or `(last_bit_indices[i] - 1)'th` bit) and
 * subtract the count from the range's null count.
 *
 * Expects `0 <= first_bit_indices[i] <= last_bit_indices[i]`.
 *
 * @param[in] bitmask The bitmask whose non-zero bits outside the range in the
 * boundary words will be counted.
 * @param[in] num_ranges The number of ranges
 * @param[in] first_bit_indices The indices (inclusive) of the first bit in each
 * range
 * @param[in] last_bit_indices The indices (exclusive) of the last bit in each
 * range
 * @param[in,out] null_counts The number of non-zero bits in each range to be
 * updated
 */
template <typename OffsetIterator, typename OutputIterator>
__global__ void subtract_set_bits_range_boundaries_kerenel(bitmask_type const *bitmask,
                                                           size_type num_ranges,
                                                           OffsetIterator first_bit_indices,
                                                           OffsetIterator last_bit_indices,
                                                           OutputIterator null_counts)
{
  constexpr size_type const word_size_in_bits{detail::size_in_bits<bitmask_type>()};

  cudf::size_type const tid = threadIdx.x + blockIdx.x * blockDim.x;
  cudf::size_type range_id  = tid;

  while (range_id < num_ranges) {
    size_type const first_bit_index = *(first_bit_indices + range_id);
    size_type const last_bit_index  = *(last_bit_indices + range_id);
    size_type delta                 = 0;
    size_type num_slack_bits        = 0;

    // compute delta due to the preceding bits in the first word in the range

    num_slack_bits = intra_word_index(first_bit_index);
    if (num_slack_bits > 0) {
      bitmask_type word       = bitmask[word_index(first_bit_index)];
      bitmask_type slack_mask = set_least_significant_bits(num_slack_bits);
      delta -= __popc(word & slack_mask);
    }

    // compute delta due to the following bits in the last word in the range

    num_slack_bits = (last_bit_index % word_size_in_bits) == 0
                       ? 0
                       : word_size_in_bits - intra_word_index(last_bit_index);
    if (num_slack_bits > 0) {
      bitmask_type word       = bitmask[word_index(last_bit_index)];
      bitmask_type slack_mask = set_most_significant_bits(num_slack_bits);
      delta -= __popc(word & slack_mask);
    }

    size_type updated_null_count = *(null_counts + range_id) + delta;
    *(null_counts + range_id)    = updated_null_count;

    range_id += blockDim.x * gridDim.x;
  }
}

/**
 * @brief Copies the bits starting at the specified offset from a source
 * bitmask into the destination bitmask.
 *
 * Bit `i` in `destination` will be equal to bit `i + offset` from `source`.
 *
 * @param destination The mask to copy into
 * @param source The mask to copy from
 * @param source_begin_bit The offset into `source` from which to begin the copy
 * @param source_end_bit   The offset into `source` till which copying is done
 * @param number_of_mask_words The number of `cudf::bitmask_type` words to copy
 */
// TODO: Also make binops test that uses offset in column_view
__global__ void copy_offset_bitmask(bitmask_type *__restrict__ destination,
                                    bitmask_type const *__restrict__ source,
                                    size_type source_begin_bit,
                                    size_type source_end_bit,
                                    size_type number_of_mask_words)
{
  for (size_type destination_word_index = threadIdx.x + blockIdx.x * blockDim.x;
       destination_word_index < number_of_mask_words;
       destination_word_index += blockDim.x * gridDim.x) {
    destination[destination_word_index] = detail::get_mask_offset_word(
      source, destination_word_index, source_begin_bit, source_end_bit);
  }
}

/**
 * @brief Computes the bitwise AND of an array of bitmasks
 *
 * @param destination The bitmask to write result into
 * @param source Array of source mask pointers. All masks must be of same size
 * @param begin_bit Array of offsets into corresponding @p source masks.
 *                  Must be same size as source array
 * @param num_sources Number of masks in @p source array
 * @param source_size Number of bits in each mask in @p source
 * @param number_of_mask_words The number of words of type bitmask_type to copy
 */
__global__ void offset_bitmask_and(bitmask_type *__restrict__ destination,
                                   bitmask_type const *const *__restrict__ source,
                                   size_type const *__restrict__ begin_bit,
                                   size_type num_sources,
                                   size_type source_size,
                                   size_type number_of_mask_words)
{
  for (size_type destination_word_index = threadIdx.x + blockIdx.x * blockDim.x;
       destination_word_index < number_of_mask_words;
       destination_word_index += blockDim.x * gridDim.x) {
    bitmask_type destination_word = ~bitmask_type{0};  // All bits 1
    for (size_type i = 0; i < num_sources; i++) {
      destination_word &= detail::get_mask_offset_word(
        source[i], destination_word_index, begin_bit[i], begin_bit[i] + source_size);
    }

    destination[destination_word_index] = destination_word;
  }
}

// convert [first_bit_index,last_bit_index) to
// [first_word_index,last_word_index)
struct to_word_index : public thrust::unary_function<size_type, size_type> {
  const bool _inclusive                 = false;
  size_type const *const _d_bit_indices = nullptr;

  /**
   * @brief Constructor of a functor that converts bit indices to bitmask word
   * indices.
   *
   * @param[in] inclusive Flag that indicates whether bit indices are inclusive
   * or exclusive.
   * @param[in] d_bit_indices Pointer to an array of bit indices
   */
  __host__ to_word_index(bool inclusive, size_type const *d_bit_indices)
    : _inclusive(inclusive), _d_bit_indices(d_bit_indices)
  {
  }

  __device__ size_type operator()(const size_type &i) const
  {
    auto bit_index = _d_bit_indices[i];
    return word_index(bit_index) + ((_inclusive || intra_word_index(bit_index) == 0) ? 0 : 1);
  }
};

}  // namespace

namespace detail {

// Create a bitmask from a specific range
rmm::device_buffer copy_bitmask(bitmask_type const *mask,
                                size_type begin_bit,
                                size_type end_bit,
                                rmm::cuda_stream_view stream,
                                rmm::mr::device_memory_resource *mr)
{
  CUDF_FUNC_RANGE();
  CUDF_EXPECTS(begin_bit >= 0, "Invalid range.");
  CUDF_EXPECTS(begin_bit <= end_bit, "Invalid bit range.");
  rmm::device_buffer dest_mask{};
  auto num_bytes = bitmask_allocation_size_bytes(end_bit - begin_bit);
  if ((mask == nullptr) || (num_bytes == 0)) { return dest_mask; }
  if (begin_bit == 0) {
    dest_mask = rmm::device_buffer{static_cast<void const *>(mask), num_bytes, stream, mr};
  } else {
    auto number_of_mask_words = num_bitmask_words(end_bit - begin_bit);
    dest_mask                 = rmm::device_buffer{num_bytes, stream, mr};
    cudf::detail::grid_1d config(number_of_mask_words, 256);
    copy_offset_bitmask<<<config.num_blocks, config.num_threads_per_block, 0, stream.value()>>>(
      static_cast<bitmask_type *>(dest_mask.data()),
      mask,
      begin_bit,
      end_bit,
      number_of_mask_words);
    CHECK_CUDA(stream.value());
  }
  return dest_mask;
}

// Create a bitmask from a column view
rmm::device_buffer copy_bitmask(column_view const &view,
                                rmm::cuda_stream_view stream,
                                rmm::mr::device_memory_resource *mr)
{
  CUDF_FUNC_RANGE();
  rmm::device_buffer null_mask{0, stream, mr};
  if (view.nullable()) {
    null_mask =
      copy_bitmask(view.null_mask(), view.offset(), view.offset() + view.size(), stream, mr);
  }
  return null_mask;
}

// Inplace Bitwise AND of the masks
void inplace_bitmask_and(bitmask_type *dest_mask,
                         std::vector<bitmask_type const *> const &masks,
                         std::vector<size_type> const &begin_bits,
                         size_type mask_size,
                         rmm::cuda_stream_view stream,
                         rmm::mr::device_memory_resource *mr)
{
  CUDF_EXPECTS(std::all_of(begin_bits.begin(), begin_bits.end(), [](auto b) { return b >= 0; }),
               "Invalid range.");
  CUDF_EXPECTS(mask_size > 0, "Invalid bit range.");
  CUDF_EXPECTS(std::all_of(masks.begin(), masks.end(), [](auto p) { return p != nullptr; }),
               "Mask pointer cannot be null");

  auto number_of_mask_words = num_bitmask_words(mask_size);

  rmm::device_vector<bitmask_type const *> d_masks(masks);
  rmm::device_vector<size_type> d_begin_bits(begin_bits);

  cudf::detail::grid_1d config(number_of_mask_words, 256);
  offset_bitmask_and<<<config.num_blocks, config.num_threads_per_block, 0, stream.value()>>>(
    dest_mask,
    d_masks.data().get(),
    d_begin_bits.data().get(),
    d_masks.size(),
    mask_size,
    number_of_mask_words);

  CHECK_CUDA(stream.value());
}

// Bitwise AND of the masks
rmm::device_buffer bitmask_and(std::vector<bitmask_type const *> const &masks,
                               std::vector<size_type> const &begin_bits,
                               size_type mask_size,
                               rmm::cuda_stream_view stream,
                               rmm::mr::device_memory_resource *mr)
{
  rmm::device_buffer dest_mask{};
  auto num_bytes = bitmask_allocation_size_bytes(mask_size);

  dest_mask = rmm::device_buffer{num_bytes, stream, mr};
  inplace_bitmask_and(
    static_cast<bitmask_type *>(dest_mask.data()), masks, begin_bits, mask_size, stream, mr);

  return dest_mask;
}

cudf::size_type count_set_bits(bitmask_type const *bitmask,
                               size_type start,
                               size_type stop,
                               rmm::cuda_stream_view stream = rmm::cuda_stream_default)
{
  if (nullptr == bitmask) { return 0; }

  CUDF_EXPECTS(start >= 0, "Invalid range.");
  CUDF_EXPECTS(start <= stop, "Invalid bit range.");

  std::size_t num_bits_to_count = stop - start;
  if (num_bits_to_count == 0) { return 0; }

  auto num_words = num_bitmask_words(num_bits_to_count);

  constexpr size_type block_size{256};

  cudf::detail::grid_1d grid(num_words, block_size);

  rmm::device_scalar<size_type> non_zero_count(0, stream);

  count_set_bits_kernel<block_size>
    <<<grid.num_blocks, grid.num_threads_per_block, 0, stream.value()>>>(
      bitmask, start, stop - 1, non_zero_count.data());

  return non_zero_count.value(stream);
}

cudf::size_type count_unset_bits(bitmask_type const *bitmask,
                                 size_type start,
                                 size_type stop,
                                 rmm::cuda_stream_view stream = rmm::cuda_stream_default)
{
  if (nullptr == bitmask) { return 0; }
  auto num_bits = (stop - start);
  return (num_bits - detail::count_set_bits(bitmask, start, stop, stream));
}

std::vector<size_type> segmented_count_set_bits(bitmask_type const *bitmask,
                                                std::vector<size_type> const &indices,
                                                rmm::cuda_stream_view stream)
{
  CUDF_EXPECTS(indices.size() % 2 == 0,
               "Array of indices needs to have an even number of elements.");
  for (size_t i = 0; i < indices.size() / 2; i++) {
    auto begin = indices[i * 2];
    auto end   = indices[i * 2 + 1];
    CUDF_EXPECTS(begin >= 0, "Starting index cannot be negative.");
    CUDF_EXPECTS(end >= begin, "End index cannot be smaller than the starting index.");
  }

  if (indices.empty()) {
    return std::vector<size_type>{};
  } else if (bitmask == nullptr) {
    std::vector<size_type> ret(indices.size() / 2);
    for (size_t i = 0; i < indices.size() / 2; i++) {
      ret[i] = indices[2 * i + 1] - indices[2 * i];
    }
    return ret;
  }

  size_type num_ranges = indices.size() / 2;
  thrust::host_vector<size_type> h_first_indices(num_ranges);
  thrust::host_vector<size_type> h_last_indices(num_ranges);
  thrust::stable_partition_copy(thrust::seq,
                                std::begin(indices),
                                std::end(indices),
                                thrust::make_counting_iterator(0),
                                h_first_indices.begin(),
                                h_last_indices.begin(),
                                [](auto i) { return (i % 2) == 0; });

  rmm::device_vector<size_type> d_first_indices = h_first_indices;
  rmm::device_vector<size_type> d_last_indices  = h_last_indices;
  rmm::device_vector<size_type> d_null_counts(num_ranges, 0);

  auto word_num_set_bits = thrust::make_transform_iterator(
    thrust::make_counting_iterator(0),
    [bitmask] __device__(auto i) { return static_cast<size_type>(__popc(bitmask[i])); });
  auto first_word_indices = thrust::make_transform_iterator(
    thrust::make_counting_iterator(0),
    // We cannot use lambda as hipcub::DeviceSegmentedReduce::Sum() requires
    // first_word_indices and last_word_indices to have the same type.
    to_word_index(true, d_first_indices.data().get()));
  auto last_word_indices = thrust::make_transform_iterator(
    thrust::make_counting_iterator(0),
    // We cannot use lambda as hipcub::DeviceSegmentedReduce::Sum() requires
    // first_word_indices and last_word_indices to have the same type.
    to_word_index(false, d_last_indices.data().get()));

  // first allocate temporary memroy

  size_t temp_storage_bytes{0};
  CUDA_TRY(hipcub::DeviceSegmentedReduce::Sum(nullptr,
                                           temp_storage_bytes,
                                           word_num_set_bits,
                                           d_null_counts.begin(),
                                           num_ranges,
                                           first_word_indices,
                                           last_word_indices,
                                           stream.value()));
  rmm::device_buffer d_temp_storage(temp_storage_bytes, stream);

  // second perform segmented reduction

  CUDA_TRY(hipcub::DeviceSegmentedReduce::Sum(d_temp_storage.data(),
                                           temp_storage_bytes,
                                           word_num_set_bits,
                                           d_null_counts.begin(),
                                           num_ranges,
                                           first_word_indices,
                                           last_word_indices,
                                           stream.value()));

  CHECK_CUDA(stream.value());

  // third, adjust counts in segment boundaries (if segments are not
  // word-aligned)

  constexpr size_type block_size{256};

  cudf::detail::grid_1d grid(num_ranges, block_size);

  subtract_set_bits_range_boundaries_kerenel<<<grid.num_blocks,
                                               grid.num_threads_per_block,
                                               0,
                                               stream.value()>>>(
    bitmask, num_ranges, d_first_indices.begin(), d_last_indices.begin(), d_null_counts.begin());

  CHECK_CUDA(stream.value());

  std::vector<size_type> ret(num_ranges);
  CUDA_TRY(hipMemcpyAsync(ret.data(),
                           d_null_counts.data().get(),
                           num_ranges * sizeof(size_type),
                           hipMemcpyDeviceToHost,
                           stream.value()));

  stream.synchronize();  // now ret is valid.

  return ret;
}

std::vector<size_type> segmented_count_unset_bits(bitmask_type const *bitmask,
                                                  std::vector<size_type> const &indices,
                                                  rmm::cuda_stream_view stream)
{
  if (indices.empty()) {
    return std::vector<size_type>{};
  } else if (bitmask == nullptr) {
    return std::vector<size_type>(indices.size() / 2, 0);
  }

  auto ret = segmented_count_set_bits(bitmask, indices, stream);
  for (size_t i = 0; i < ret.size(); i++) {
    auto begin = indices[i * 2];
    auto end   = indices[i * 2 + 1];
    ret[i]     = (end - begin) - ret[i];
  }

  return ret;
}

// Returns the bitwise AND of the null masks of all columns in the table view
rmm::device_buffer bitmask_and(table_view const &view,
                               rmm::cuda_stream_view stream,
                               rmm::mr::device_memory_resource *mr)
{
  CUDF_FUNC_RANGE();
  rmm::device_buffer null_mask{0, stream, mr};
  if (view.num_rows() == 0 or view.num_columns() == 0) { return null_mask; }

  std::vector<bitmask_type const *> masks;
  std::vector<size_type> offsets;
  for (auto &&col : view) {
    if (col.nullable()) {
      masks.push_back(col.null_mask());
      offsets.push_back(col.offset());
    }
  }

  if (masks.size() > 0) {
    return cudf::detail::bitmask_and(masks, offsets, view.num_rows(), stream, mr);
  }

  return null_mask;
}

}  // namespace detail

// Count non-zero bits in the specified range
cudf::size_type count_set_bits(bitmask_type const *bitmask, size_type start, size_type stop)
{
  CUDF_FUNC_RANGE();
  return detail::count_set_bits(bitmask, start, stop);
}

// Count zero bits in the specified range
cudf::size_type count_unset_bits(bitmask_type const *bitmask, size_type start, size_type stop)
{
  CUDF_FUNC_RANGE();
  return detail::count_unset_bits(bitmask, start, stop);
}

// Count non-zero bits in the specified ranges
std::vector<size_type> segmented_count_set_bits(bitmask_type const *bitmask,
                                                std::vector<size_type> const &indices)
{
  CUDF_FUNC_RANGE();
  return detail::segmented_count_set_bits(bitmask, indices, rmm::cuda_stream_default);
}

// Count zero bits in the specified ranges
std::vector<size_type> segmented_count_unset_bits(bitmask_type const *bitmask,
                                                  std::vector<size_type> const &indices)
{
  CUDF_FUNC_RANGE();
  return detail::segmented_count_unset_bits(bitmask, indices, rmm::cuda_stream_default);
}

// Create a bitmask from a specific range
rmm::device_buffer copy_bitmask(bitmask_type const *mask,
                                size_type begin_bit,
                                size_type end_bit,
                                rmm::mr::device_memory_resource *mr)
{
  return detail::copy_bitmask(mask, begin_bit, end_bit, rmm::cuda_stream_default, mr);
}

// Create a bitmask from a column view
rmm::device_buffer copy_bitmask(column_view const &view, rmm::mr::device_memory_resource *mr)
{
  return detail::copy_bitmask(view, rmm::cuda_stream_default, mr);
}

rmm::device_buffer bitmask_and(table_view const &view, rmm::mr::device_memory_resource *mr)
{
  return detail::bitmask_and(view, rmm::cuda_stream_default, mr);
}

}  // namespace cudf
