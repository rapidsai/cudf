#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <cudf/null_mask.hpp>
#include <cudf/utilities/bit.hpp>
#include <cudf/utilities/error.hpp>
#include <cudf/detail/utilities/integer_utils.hpp>
#include <utilities/cuda_utils.hpp>


#include <thrust/copy.h>
#include <thrust/device_ptr.h>
#include <thrust/extrema.h>
#include <hipcub/hipcub.hpp>
#include <rmm/device_buffer.hpp>
#include <rmm/device_scalar.hpp>
#include <rmm/mr/device_memory_resource.hpp>

namespace cudf {

size_type state_null_count(mask_state state, size_type size) {
  switch (state) {
    case UNALLOCATED:
      return 0;
    case UNINITIALIZED:
      return UNKNOWN_NULL_COUNT;
    case ALL_NULL:
      return size;
    case ALL_VALID:
      return 0;
    default:
      CUDF_FAIL("Invalid null mask state.");
  }
}

// Computes required allocation size of a bitmask
std::size_t bitmask_allocation_size_bytes(size_type number_of_bits,
                                          std::size_t padding_boundary) {
  CUDF_EXPECTS(padding_boundary > 0, "Invalid padding boundary");
  auto necessary_bytes =
      cudf::util::div_rounding_up_safe<size_type>(number_of_bits, CHAR_BIT);

  auto padded_bytes =
      padding_boundary * cudf::util::div_rounding_up_safe<size_type>(
                             necessary_bytes, padding_boundary);
  return padded_bytes;
}

// Computes number of *actual* bitmask_type elements needed
size_type num_bitmask_words(size_type number_of_bits) {
  return cudf::util::div_rounding_up_safe<size_type>(
      number_of_bits, detail::size_in_bits<bitmask_type>());
}

// Create a device_buffer for a null mask
rmm::device_buffer create_null_mask(size_type size, mask_state state,
                                    hipStream_t stream,
                                    rmm::mr::device_memory_resource *mr) {
  size_type mask_size{0};

  if (state != UNALLOCATED) {
    mask_size = bitmask_allocation_size_bytes(size);
  }

  rmm::device_buffer mask(mask_size, stream, mr);

  if (state != UNINITIALIZED) {
    uint8_t fill_value = (state == ALL_VALID) ? 0xff : 0x00;
    CUDA_TRY(hipMemsetAsync(static_cast<bitmask_type *>(mask.data()),
                             fill_value, mask_size, stream));
  }

  return mask;
}

namespace {

/**---------------------------------------------------------------------------*
 * @brief Counts the number of non-zero bits in a bitmask in the range
 * `[first_bit_index, last_bit_index]`.
 *
 * Expects `0 <= first_bit_index <= last_bit_index`.
 *
 * @param[in] bitmask The bitmask whose non-zero bits will be counted.
 * @param[in] first_bit_index The index (inclusive) of the first bit to count
 * @param[in] last_bit_index The index (inclusive) of the last bit to count
 * @param[out] global_count The number of non-zero bits in the specified range
 *---------------------------------------------------------------------------**/
template <size_type block_size>
__global__ void count_set_bits_kernel(bitmask_type const *bitmask,
                                      size_type first_bit_index,
                                      size_type last_bit_index,
                                      size_type *global_count) {
  constexpr auto const word_size{detail::size_in_bits<bitmask_type>()};

  auto const first_word_index{word_index(first_bit_index)};
  auto const last_word_index{word_index(last_bit_index)};
  auto const tid = threadIdx.x + blockIdx.x * blockDim.x;
  auto thread_word_index = tid + first_word_index;
  size_type thread_count{0};

  // First, just count the bits in all words
  while (thread_word_index <= last_word_index) {
    thread_count += __popc(bitmask[thread_word_index]);
    thread_word_index += blockDim.x * gridDim.x;
  }

  // Subtract any slack bits counted from the first and last word
  // Two threads handle this -- one for first word, one for last
  if (tid < 2) {
    bool const first{tid == 0};
    bool const last{not first};

    size_type bit_index = (first) ? first_bit_index : last_bit_index;
    size_type word_index = (first) ? first_word_index : last_word_index;

    size_type num_slack_bits = bit_index % word_size;
    if (last) {
      num_slack_bits = word_size - num_slack_bits - 1;
    }

    if (num_slack_bits > 0) {
      bitmask_type word = bitmask[word_index];
      auto slack_mask = (first) ? set_least_significant_bits(num_slack_bits)
                                : set_most_significant_bits(num_slack_bits);

      thread_count -= __popc(word & slack_mask);
    }
  }

  using BlockReduce = hipcub::BlockReduce<size_type, block_size>;
  __shared__ typename BlockReduce::TempStorage temp_storage;
  size_type block_count{BlockReduce(temp_storage).Sum(thread_count)};

  if (threadIdx.x == 0) {
    atomicAdd(global_count, block_count);
  }
}

/**---------------------------------------------------------------------------*
 * @brief Copies the bits starting at the specified offset from a source
 * bitmask into the destination bitmask.
 *
 * Bit `i` in `destination` will be equal to bit `i + offset` from `source`.
 *
 * @param destination The mask to copy into
 * @param source The mask to copy from
 * @param bit_offset The offset into `source` from which to begin the copy
 * @param number_of_mask_words The number of words of type bitmask_type to copy
 *---------------------------------------------------------------------------**/
__global__ void copy_offset_bitmask(bitmask_type *__restrict__ destination,
                                    bitmask_type const *__restrict__ source,
                                    size_type bit_offset,
                                    size_type number_of_mask_words) {
  for (size_type destination_word_index = threadIdx.x + blockIdx.x * blockDim.x;
       destination_word_index < number_of_mask_words;
       destination_word_index += blockDim.x * gridDim.x) {
    size_type source_word_index =
        destination_word_index + word_index(bit_offset);
    bitmask_type curr_word = source[source_word_index];
    bitmask_type next_word = 0;
    if (destination_word_index + 1 < number_of_mask_words) {
      next_word = source[source_word_index + 1];
    }
    bitmask_type write_word = __funnelshift_r(curr_word, next_word, bit_offset);
    destination[destination_word_index] = write_word;
  }
}

}  // namespace

namespace detail {
cudf::size_type count_set_bits(bitmask_type const *bitmask, size_type start,
                               size_type stop, hipStream_t stream = 0) {
  if (nullptr == bitmask) {
    return 0;
  }

  CUDF_EXPECTS(start >= 0, "Invalid range.");
  CUDF_EXPECTS(start <= stop, "Invalid bit range.");

  std::size_t num_bits_to_count = stop - start;
  if (num_bits_to_count == 0) {
    return 0;
  }

  auto num_words = cudf::util::div_rounding_up_safe(
      num_bits_to_count, detail::size_in_bits<bitmask_type>());

  constexpr size_type block_size{256};

  cudf::util::cuda::grid_config_1d grid(num_words, block_size);

  rmm::device_scalar<size_type> non_zero_count(0, stream);

  count_set_bits_kernel<block_size>
      <<<grid.num_blocks, grid.num_threads_per_block, 0, stream>>>(
          bitmask, start, stop - 1, non_zero_count.data());

  return non_zero_count.value();
}

cudf::size_type count_unset_bits(bitmask_type const *bitmask, size_type start,
                                 size_type stop, hipStream_t stream = 0) {
  if (nullptr == bitmask) {
    return 0;
  }
  auto num_bits = (stop - start);
  return (num_bits - detail::count_set_bits(bitmask, start, stop, stream));
}

}  // namespace detail

// Count non-zero bits in the specified range
cudf::size_type count_set_bits(bitmask_type const *bitmask, size_type start,
                               size_type stop) {
  return detail::count_set_bits(bitmask, start, stop);
}

// Count zero bits in the specified range
cudf::size_type count_unset_bits(bitmask_type const *bitmask, size_type start,
                                 size_type stop) {
  return detail::count_unset_bits(bitmask, start, stop);
}

// Create a bitmask from a specific range
rmm::device_buffer copy_bitmask(bitmask_type const *mask, size_type begin_bit,
                                size_type end_bit, hipStream_t stream,
                                rmm::mr::device_memory_resource *mr) {
  CUDF_EXPECTS(begin_bit >= 0, "Invalid range.");
  CUDF_EXPECTS(begin_bit <= end_bit, "Invalid bit range.");
  rmm::device_buffer dest_mask{};
  auto num_bytes = bitmask_allocation_size_bytes(end_bit - begin_bit);
  if ((mask == nullptr) || (num_bytes == 0)) {
    return dest_mask;
  }
  if (begin_bit == 0) {
    dest_mask = rmm::device_buffer{static_cast<void const *>(mask), num_bytes,
                                   stream, mr};
  } else {
    auto number_of_mask_words = cudf::util::div_rounding_up_safe(
        static_cast<size_t>(end_bit - begin_bit),
        detail::size_in_bits<bitmask_type>());
    dest_mask = rmm::device_buffer{num_bytes, stream, mr};
    cudf::util::cuda::grid_config_1d config(number_of_mask_words, 256);
    copy_offset_bitmask<<<config.num_blocks, config.num_threads_per_block, 0,
                          stream>>>(
        static_cast<bitmask_type *>(dest_mask.data()), mask, begin_bit,
        number_of_mask_words);
    CUDA_CHECK_LAST()
  }
  return dest_mask;
}

// Create a bitmask from a specific range
rmm::device_buffer copy_bitmask(column_view const &view, hipStream_t stream,
                                rmm::mr::device_memory_resource *mr) {
  rmm::device_buffer null_mask{};
  if (view.nullable()) {
    null_mask = copy_bitmask(view.null_mask(), view.offset(),
                             view.offset() + view.size(), stream, mr);
  }
  return null_mask;
}

}  // namespace cudf
