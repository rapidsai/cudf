/*
 * Copyright (c) 2020-2021, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <cudf/detail/aggregation/aggregation.cuh>

#include <rmm/cuda_stream_view.hpp>

namespace cudf {
namespace detail {
void initialize_with_identity(mutable_table_view& table,
                              std::vector<aggregation::Kind> const& aggs,
                              rmm::cuda_stream_view stream)
{
  // TODO: Initialize all the columns in a single kernel instead of invoking one
  // kernel per column
  for (size_type i = 0; i < table.num_columns(); ++i) {
    auto col        = table.column(i);
    auto const type = data_type{device_storage_type_id(col.type().id())};
    dispatch_type_and_aggregation(type, aggs[i], identity_initializer{}, col, stream);
  }
}

}  // namespace detail
}  // namespace cudf
