#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2021, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <cstddef>
#include <cstdint>
#include <transform/jit/operation-udf.hpp>

#include <cudf/types.hpp>
#include <cudf/utilities/bit.hpp>

#include <cuda/std/climits>
#include <cuda/std/cstddef>
#include <cuda/std/limits>
#include <cuda/std/tuple>
#include <cuda/std/type_traits>

namespace cudf {
namespace transformation {
namespace jit {

template <typename T>
struct Masked {
  T value;
  bool valid;
};

template <typename TypeIn, typename MaskType, typename OffsetType>
__device__ auto make_args(cudf::size_type id, TypeIn in_ptr, MaskType in_mask, OffsetType in_offset)
{
  bool valid = in_mask ? cudf::bit_is_set(in_mask, in_offset + id) : true;
  return cuda::std::make_tuple(in_ptr[id], valid);
}

template <typename InType, typename MaskType, typename OffsetType, typename... Arguments>
__device__ auto make_args(cudf::size_type id,
                          InType in_ptr,
                          MaskType in_mask,      // in practice, always cudf::bitmask_type const*
                          OffsetType in_offset,  // in practice, always cudf::size_type
                          Arguments... args)
{
  bool valid = in_mask ? cudf::bit_is_set(in_mask, in_offset + id) : true;
  return cuda::std::tuple_cat(cuda::std::make_tuple(in_ptr[id], valid), make_args(id, args...));
}

template <typename TypeOut, typename... Arguments>
__global__ void generic_udf_kernel(cudf::size_type size,
                                   TypeOut* out_data,
                                   bool* out_mask,
                                   Arguments... args)
{
  int const tid    = threadIdx.x;
  int const blkid  = blockIdx.x;
  int const blksz  = blockDim.x;
  int const gridsz = gridDim.x;
  int const start  = tid + blkid * blksz;
  int const step   = blksz * gridsz;

  Masked<TypeOut> output;
  for (cudf::size_type i = start; i < size; i += step) {
    auto func_args = cuda::std::tuple_cat(
      cuda::std::make_tuple(&output.value),
      make_args(i, args...)  // passed int64*, bool*, int64, int64*, bool*, int64
    );
    cuda::std::apply(GENERIC_OP, func_args);
    out_data[i] = output.value;
    out_mask[i] = output.valid;
  }
}

}  // namespace jit
}  // namespace transformation
}  // namespace cudf
