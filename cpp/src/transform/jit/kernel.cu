/*
 * Copyright (c) 2019-2025, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <cudf/ast/detail/operator_functor.cuh>
#include <cudf/column/column_device_view_base.cuh>
#include <cudf/detail/utilities/grid_1d.cuh>
#include <cudf/strings/string_view.cuh>
#include <cudf/types.hpp>
#include <cudf/wrappers/durations.hpp>
#include <cudf/wrappers/timestamps.hpp>

#include <cuda/std/cstddef>

#include <jit/accessors.cuh>
#include <jit/span.cuh>

// clang-format off
// This header is an inlined header that defines the GENERIC_FILTER_OP function. It is placed here
// so the symbols in the headers above can be used by it.
#include <cudf/detail/operation-udf.hpp>
// clang-format on

namespace cudf {
namespace transformation {
namespace jit {

template <null_aware is_null_aware,
          bool may_evaluate_null,
          bool has_user_data,
          typename Out,
          typename... In>
CUDF_KERNEL void kernel(cudf::mutable_column_device_view_core const* outputs,
                        cudf::column_device_view_core const* inputs,
                        bool* null_mask,
                        void* user_data)
{
  // inputs to JITIFY kernels have to be either sized-integral types or pointers. Structs or
  // references can't be passed directly/correctly as they will be crossing an ABI boundary

  auto const start  = cudf::detail::grid_1d::global_thread_id();
  auto const stride = cudf::detail::grid_1d::grid_stride();
  auto const size   = outputs[0].size();

  for (auto i = start; i < size; i += stride) {
    if constexpr (is_null_aware == null_aware::NO) {
      auto const is_valid = (true && ... && In::is_valid(inputs, i));
      if constexpr (may_evaluate_null) { null_mask[i] = is_valid; }

      if (!is_valid) { continue; }

      if constexpr (has_user_data) {
        GENERIC_TRANSFORM_OP(user_data, i, &Out::element(outputs, i), In::element(inputs, i)...);
      } else {
        GENERIC_TRANSFORM_OP(&Out::element(outputs, i), In::element(inputs, i)...);
      }

    } else {  // is_null_aware == null_aware::YES
      cuda::std::optional<typename Out::type> result;

      if constexpr (has_user_data) {
        GENERIC_TRANSFORM_OP(user_data, i, &result, In::nullable_element(inputs, i)...);
      } else {
        GENERIC_TRANSFORM_OP(&result, In::nullable_element(inputs, i)...);
      }

      Out::assign(outputs, i, *result);

      if constexpr (may_evaluate_null) { null_mask[i] = result.has_value(); }
    }
  }
}

template <null_aware is_null_aware,
          bool may_evaluate_null,
          bool has_user_data,
          typename Out,
          typename... In>
CUDF_KERNEL void fixed_point_kernel(cudf::mutable_column_device_view_core const* outputs,
                                    cudf::column_device_view_core const* inputs,
                                    bool* null_mask,
                                    void* user_data)
{
  auto const start        = cudf::detail::grid_1d::global_thread_id();
  auto const stride       = cudf::detail::grid_1d::grid_stride();
  auto const size         = outputs[0].size();
  auto const output_scale = static_cast<numeric::scale_type>(outputs[0].type().scale());

  for (auto i = start; i < size; i += stride) {
    if constexpr (is_null_aware == null_aware::NO) {
      auto const is_valid = (true && ... && In::is_valid(inputs, i));

      if constexpr (may_evaluate_null) { null_mask[i] = is_valid; }

      if (!is_valid) { continue; }

      typename Out::type result{numeric::scaled_integer<typename Out::type::rep>{0, output_scale}};

      if constexpr (has_user_data) {
        GENERIC_TRANSFORM_OP(user_data, i, &result, In::element(inputs, i)...);
      } else {
        GENERIC_TRANSFORM_OP(&result, In::element(inputs, i)...);
      }

      Out::assign(outputs, i, result);

    } else {  // is_null_aware == null_aware::YES
      cuda::std::optional<typename Out::type> result{
        typename Out::type{numeric::scaled_integer<typename Out::type::rep>{0, output_scale}}};

      if constexpr (has_user_data) {
        GENERIC_TRANSFORM_OP(user_data, i, &result, In::nullable_element(inputs, i)...);
      } else {
        GENERIC_TRANSFORM_OP(&result, In::nullable_element(inputs, i)...);
      }

      Out::assign(outputs, i, *result);

      if constexpr (may_evaluate_null) { null_mask[i] = result.has_value(); }
    }
  }
}

template <null_aware is_null_aware,
          bool may_evaluate_null,
          bool has_user_data,
          typename Out,
          typename... In>
CUDF_KERNEL void span_kernel(cudf::jit::device_optional_span<typename Out::type> const* outputs,
                             cudf::column_device_view_core const* inputs,
                             bool* null_mask,
                             void* user_data)
{
  auto const start  = cudf::detail::grid_1d::global_thread_id();
  auto const stride = cudf::detail::grid_1d::grid_stride();
  auto const size   = outputs[0].size();

  for (auto i = start; i < size; i += stride) {
    if constexpr (is_null_aware == null_aware::NO) {
      auto const is_valid = (true && ... && In::is_valid(inputs, i));

      if constexpr (may_evaluate_null) { null_mask[i] = is_valid; }

      if (!is_valid) { continue; }

      if constexpr (has_user_data) {
        GENERIC_TRANSFORM_OP(user_data, i, &Out::element(outputs, i), In::element(inputs, i)...);
      } else {
        GENERIC_TRANSFORM_OP(&Out::element(outputs, i), In::element(inputs, i)...);
      }
    } else {  // is_null_aware == null_aware::YES
      cuda::std::optional<typename Out::type> result;

      if constexpr (has_user_data) {
        GENERIC_TRANSFORM_OP(user_data, i, &result, In::nullable_element(inputs, i)...);
      } else {
        GENERIC_TRANSFORM_OP(&result, In::nullable_element(inputs, i)...);
      }

      Out::assign(outputs, i, *result);

      if constexpr (may_evaluate_null) { null_mask[i] = result.has_value(); }
    }
  }
}

}  // namespace jit
}  // namespace transformation
}  // namespace cudf
