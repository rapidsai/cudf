#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <cudf/column/column_device_view.cuh>
#include <cudf/column/column_view.hpp>
#include <cudf/detail/nvtx/ranges.hpp>
#include <cudf/detail/transform.hpp>
#include <cudf/detail/valid_if.cuh>
#include <cudf/null_mask.hpp>
#include <cudf/types.hpp>
#include <cudf/utilities/traits.hpp>
#include <cudf/utilities/type_dispatcher.hpp>

namespace cudf {
namespace detail {
struct dispatch_nan_to_null {
  template <typename T>
  std::enable_if_t<std::is_floating_point<T>::value,
                   std::pair<std::unique_ptr<rmm::device_buffer>, cudf::size_type>>
  operator()(column_view const& input, rmm::mr::device_memory_resource* mr, hipStream_t stream)
  {
    auto input_device_view_ptr = column_device_view::create(input, stream);
    auto input_device_view     = *input_device_view_ptr;

    if (input.nullable()) {
      auto pred = [input_device_view] __device__(cudf::size_type idx) {
        return not(std::isnan(input_device_view.element<T>(idx)) ||
                   input_device_view.is_null_nocheck(idx));
      };

      auto mask = detail::valid_if(thrust::make_counting_iterator<cudf::size_type>(0),
                                   thrust::make_counting_iterator<cudf::size_type>(input.size()),
                                   pred,
                                   stream,
                                   mr);

      return std::make_pair(std::make_unique<rmm::device_buffer>(std::move(mask.first)),
                            mask.second);
    } else {
      auto pred = [input_device_view] __device__(cudf::size_type idx) {
        return not(std::isnan(input_device_view.element<T>(idx)));
      };

      auto mask = detail::valid_if(thrust::make_counting_iterator<cudf::size_type>(0),
                                   thrust::make_counting_iterator<cudf::size_type>(input.size()),
                                   pred,
                                   stream,
                                   mr);

      return std::make_pair(std::make_unique<rmm::device_buffer>(std::move(mask.first)),
                            mask.second);
    }
  }

  template <typename T>
  std::enable_if_t<!std::is_floating_point<T>::value,
                   std::pair<std::unique_ptr<rmm::device_buffer>, cudf::size_type>>
  operator()(column_view const& input, rmm::mr::device_memory_resource* mr, hipStream_t stream)
  {
    CUDF_FAIL("Input column can't be a non-floating type");
  }
};

std::pair<std::unique_ptr<rmm::device_buffer>, cudf::size_type> nans_to_nulls(
  column_view const& input, rmm::mr::device_memory_resource* mr, hipStream_t stream)
{
  if (input.is_empty()) { return std::make_pair(std::make_unique<rmm::device_buffer>(), 0); }

  return cudf::type_dispatcher(input.type(), dispatch_nan_to_null{}, input, mr, stream);
}

}  // namespace detail

std::pair<std::unique_ptr<rmm::device_buffer>, cudf::size_type> nans_to_nulls(
  column_view const& input, rmm::mr::device_memory_resource* mr)
{
  CUDF_FUNC_RANGE();
  return detail::nans_to_nulls(input, mr);
}

}  // namespace cudf
