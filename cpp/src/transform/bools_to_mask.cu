/*
 * Copyright (c) 2019, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <cudf/column/column_device_view.cuh>
#include <cudf/column/column_view.hpp>
#include <cudf/detail/iterator.cuh>
#include <cudf/detail/nvtx/ranges.hpp>
#include <cudf/detail/transform.hpp>
#include <cudf/detail/valid_if.cuh>
#include <cudf/null_mask.hpp>
#include <cudf/types.hpp>
#include <cudf/utilities/traits.hpp>
#include <cudf/utilities/type_dispatcher.hpp>

namespace cudf {
namespace detail {
std::pair<std::unique_ptr<rmm::device_buffer>, cudf::size_type> bools_to_mask(
  column_view const& input, rmm::mr::device_memory_resource* mr, hipStream_t stream)
{
  CUDF_EXPECTS(input.type().id() == type_id::BOOL8, "Input is not of type bool");

  if (input.size() == 0) { return std::make_pair(std::make_unique<rmm::device_buffer>(), 0); }

  auto input_device_view_ptr = column_device_view::create(input, stream);
  auto input_device_view     = *input_device_view_ptr;
  auto pred                  = [] __device__(bool element) { return element; };
  if (input.nullable()) {
    // Nulls are considered false
    auto input_begin = make_null_replacement_iterator<bool>(input_device_view, false);

    auto mask = detail::valid_if(input_begin, input_begin + input.size(), pred, stream, mr);

    return std::make_pair(std::make_unique<rmm::device_buffer>(std::move(mask.first)), mask.second);
  } else {
    auto mask = detail::valid_if(
      input_device_view.begin<bool>(), input_device_view.end<bool>(), pred, stream, mr);

    return std::make_pair(std::make_unique<rmm::device_buffer>(std::move(mask.first)), mask.second);
  }
}

}  // namespace detail

std::pair<std::unique_ptr<rmm::device_buffer>, cudf::size_type> bools_to_mask(
  column_view const& input, rmm::mr::device_memory_resource* mr)
{
  CUDF_FUNC_RANGE();
  return detail::bools_to_mask(input, mr);
}

}  // namespace cudf
