/*
 * Copyright (c) 2020, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <thrust/iterator/constant_iterator.h>
#include <thrust/transform.h>
#include <cudf/column/column_factories.hpp>
#include <cudf/column/column_view.hpp>
#include <cudf/detail/nvtx/ranges.hpp>
#include <cudf/transform.hpp>
#include <cudf/types.hpp>
#include <cudf/utilities/bit.hpp>

namespace cudf {
namespace detail {
std::unique_ptr<column> mask_to_bools(bitmask_type const* bitmask,
                                      size_type begin_bit,
                                      size_type end_bit,
                                      hipStream_t stream,
                                      rmm::mr::device_memory_resource* mr)
{
  auto const length = end_bit - begin_bit;
  CUDF_EXPECTS(length >= 0, "begin_bit should be less than or equal to end_bit");
  CUDF_EXPECTS((bitmask != nullptr) or (length == 0), "nullmask is null");

  auto out_col =
    make_fixed_width_column(data_type(type_id::BOOL8), length, mask_state::UNALLOCATED, stream, mr);

  if (length > 0) {
    auto mutable_view = out_col->mutable_view();

    thrust::transform(rmm::exec_policy(stream)->on(stream),
                      thrust::make_counting_iterator<cudf::size_type>(begin_bit),
                      thrust::make_counting_iterator<cudf::size_type>(end_bit),
                      mutable_view.begin<bool>(),
                      [bitmask] __device__(auto index) { return bit_is_set(bitmask, index); });
  }

  return out_col;
}
}  // namespace detail

std::unique_ptr<column> mask_to_bools(bitmask_type const* bitmask,
                                      size_type begin_bit,
                                      size_type end_bit,
                                      rmm::mr::device_memory_resource* mr)
{
  return detail::mask_to_bools(bitmask, begin_bit, end_bit, 0, mr);
}
}  // namespace cudf
