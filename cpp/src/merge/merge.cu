#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2020, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */
#include <cudf/copying.hpp>
#include <cudf/detail/merge.cuh>
#include <cudf/detail/nvtx/ranges.hpp>
#include <cudf/detail/utilities/cuda.cuh>
#include <cudf/dictionary/detail/merge.hpp>
#include <cudf/dictionary/detail/update_keys.hpp>
#include <cudf/strings/detail/merge.cuh>
#include <cudf/table/table.hpp>
#include <cudf/table/table_device_view.cuh>

#include <rmm/thrust_rmm_allocator.h>

#include <thrust/iterator/constant_iterator.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/merge.h>
#include <thrust/tuple.h>

#include <queue>
#include <vector>

namespace cudf {
namespace detail {
namespace {

using detail::side;
using index_type = detail::index_type;

/**
 * @brief Merges the bits of two validity bitmasks.
 *
 * Merges the bits from two column_device_views into the destination column_device_view
 * according to `merged_indices` map such that bit `i` in `out_col`
 * will be equal to bit `thrust::get<1>(merged_indices[i])` from `left_dcol`
 * if `thrust::get<0>(merged_indices[i])` equals `side::LEFT`; otherwise,
 * from `right_dcol`.
 *
 * `left_dcol`, `right_dcol` and `out_dcol` must not
 * overlap.
 *
 * @tparam left_have_valids Indicates whether left_dcol mask is unallocated (hence, ALL_VALID)
 * @tparam right_have_valids Indicates whether right_dcol mask is unallocated (hence ALL_VALID)
 * @param[in] left_dcol The left column_device_view whose bits will be merged
 * @param[in] right_dcol The right column_device_view whose bits will be merged
 * @param[out] out_dcol The output mutable_column_device_view after merging the left and right
 * @param[in] num_destination_rows The number of rows in the out_dcol
 * @param[in] merged_indices The map that indicates the source of the input and index
 * to be copied to the output. Length must be equal to `num_destination_rows`
 */
template <bool left_have_valids, bool right_have_valids>
__global__ void materialize_merged_bitmask_kernel(
  column_device_view left_dcol,
  column_device_view right_dcol,
  mutable_column_device_view out_dcol,
  size_type const num_destination_rows,
  index_type const* const __restrict__ merged_indices)
{
  size_type destination_row = threadIdx.x + blockIdx.x * blockDim.x;

  auto active_threads = __ballot_sync(0xffffffff, destination_row < num_destination_rows);

  while (destination_row < num_destination_rows) {
    index_type const& merged_idx = merged_indices[destination_row];
    side const src_side          = thrust::get<0>(merged_idx);
    size_type const src_row      = thrust::get<1>(merged_idx);
    bool const from_left{src_side == side::LEFT};
    bool source_bit_is_valid{true};
    if (left_have_valids && from_left) {
      source_bit_is_valid = left_dcol.is_valid_nocheck(src_row);
    } else if (right_have_valids && !from_left) {
      source_bit_is_valid = right_dcol.is_valid_nocheck(src_row);
    }

    // Use ballot to find all valid bits in this warp and create the output
    // bitmask element
    bitmask_type const result_mask{__ballot_sync(active_threads, source_bit_is_valid)};

    size_type const output_element = word_index(destination_row);

    // Only one thread writes output
    if (0 == threadIdx.x % warpSize) { out_dcol.set_mask_word(output_element, result_mask); }

    destination_row += blockDim.x * gridDim.x;
    active_threads = __ballot_sync(active_threads, destination_row < num_destination_rows);
  }
}

void materialize_bitmask(column_view const& left_col,
                         column_view const& right_col,
                         mutable_column_view& out_col,
                         index_type const* merged_indices,
                         hipStream_t stream)
{
  constexpr size_type BLOCK_SIZE{256};
  detail::grid_1d grid_config{out_col.size(), BLOCK_SIZE};

  auto p_left_dcol  = column_device_view::create(left_col);
  auto p_right_dcol = column_device_view::create(right_col);
  auto p_out_dcol   = mutable_column_device_view::create(out_col);

  auto left_valid  = *p_left_dcol;
  auto right_valid = *p_right_dcol;
  auto out_valid   = *p_out_dcol;

  if (left_col.has_nulls()) {
    if (right_col.has_nulls()) {
      materialize_merged_bitmask_kernel<true, true>
        <<<grid_config.num_blocks, grid_config.num_threads_per_block, 0, stream>>>(
          left_valid, right_valid, out_valid, out_col.size(), merged_indices);
    } else {
      materialize_merged_bitmask_kernel<true, false>
        <<<grid_config.num_blocks, grid_config.num_threads_per_block, 0, stream>>>(
          left_valid, right_valid, out_valid, out_col.size(), merged_indices);
    }
  } else {
    if (right_col.has_nulls()) {
      materialize_merged_bitmask_kernel<false, true>
        <<<grid_config.num_blocks, grid_config.num_threads_per_block, 0, stream>>>(
          left_valid, right_valid, out_valid, out_col.size(), merged_indices);
    } else {
      CUDF_FAIL("materialize_merged_bitmask_kernel<false, false>() should never be called.");
    }
  }

  CHECK_CUDA(stream);
}

/**
 * @brief Generates the row indices and source side (left or right) in accordance with the index
 * columns.
 *
 *
 * @tparam index_type Indicates the type to be used to collect index and side information;
 * @param[in] left_table The left table_view to be merged
 * @param[in] right_table The right table_view to be merged
 * @param[in] column_order Sort order types of index columns
 * @param[in] null_precedence Array indicating the order of nulls with respect to non-nulls for the
 * index columns
 * @param[in] nullable Flag indicating if at least one of the table_view arguments has nulls
 * (defaults to true)
 * @param[in] stream CUDA stream used for device memory operations and kernel launches.
 *
 * @return A vector of merged indices
 */
rmm::device_vector<index_type> generate_merged_indices(
  table_view const& left_table,
  table_view const& right_table,
  std::vector<order> const& column_order,
  std::vector<null_order> const& null_precedence,
  bool nullable       = true,
  hipStream_t stream = nullptr)
{
  const size_type left_size  = left_table.num_rows();
  const size_type right_size = right_table.num_rows();
  const size_type total_size = left_size + right_size;

  thrust::constant_iterator<side> left_side(side::LEFT);
  thrust::constant_iterator<side> right_side(side::RIGHT);

  auto left_indices  = thrust::make_counting_iterator(static_cast<size_type>(0));
  auto right_indices = thrust::make_counting_iterator(static_cast<size_type>(0));

  auto left_begin_zip_iterator =
    thrust::make_zip_iterator(thrust::make_tuple(left_side, left_indices));
  auto right_begin_zip_iterator =
    thrust::make_zip_iterator(thrust::make_tuple(right_side, right_indices));

  auto left_end_zip_iterator =
    thrust::make_zip_iterator(thrust::make_tuple(left_side + left_size, left_indices + left_size));
  auto right_end_zip_iterator = thrust::make_zip_iterator(
    thrust::make_tuple(right_side + right_size, right_indices + right_size));

  rmm::device_vector<index_type> merged_indices(total_size);

  auto lhs_device_view = table_device_view::create(left_table, stream);
  auto rhs_device_view = table_device_view::create(right_table, stream);

  rmm::device_vector<order> d_column_order(column_order);

  auto exec_pol = rmm::exec_policy(stream);
  if (nullable) {
    rmm::device_vector<null_order> d_null_precedence(null_precedence);

    auto ineq_op =
      detail::row_lexicographic_tagged_comparator<true>(*lhs_device_view,
                                                        *rhs_device_view,
                                                        d_column_order.data().get(),
                                                        d_null_precedence.data().get());
    thrust::merge(exec_pol->on(stream),
                  left_begin_zip_iterator,
                  left_end_zip_iterator,
                  right_begin_zip_iterator,
                  right_end_zip_iterator,
                  merged_indices.begin(),
                  ineq_op);
  } else {
    auto ineq_op = detail::row_lexicographic_tagged_comparator<false>(
      *lhs_device_view, *rhs_device_view, d_column_order.data().get());
    thrust::merge(exec_pol->on(stream),
                  left_begin_zip_iterator,
                  left_end_zip_iterator,
                  right_begin_zip_iterator,
                  right_end_zip_iterator,
                  merged_indices.begin(),
                  ineq_op);
  }

  CHECK_CUDA(stream);

  return merged_indices;
}

}  // namespace

/**
 * @brief Generate merged column given row-order of merged tables
 *  (ordered according to indices of key_cols) and the 2 columns to merge.
 */
struct column_merger {
  explicit column_merger(
    index_vector const& row_order,
    rmm::mr::device_memory_resource* mr = rmm::mr::get_current_device_resource(),
    hipStream_t stream                 = nullptr)
    : row_order_(row_order), mr_(mr), stream_(stream)
  {
  }

  // column merger operator;
  //
  template <typename Element>  // required: column type
  std::unique_ptr<column> operator()(column_view const& lcol, column_view const& rcol) const
  {
    auto lsz         = lcol.size();
    auto merged_size = lsz + rcol.size();
    auto type        = lcol.type();
    auto merged_col  = lcol.has_nulls() ? cudf::allocate_like(lcol, merged_size)
                                       : cudf::allocate_like(rcol, merged_size);

    //"gather" data from lcol, rcol according to row_order_ "map"
    //(directly calling gather() won't work because
    // lcol, rcol indices overlap!)
    //
    cudf::mutable_column_view merged_view = merged_col->mutable_view();

    // initialize null_mask to all valid:
    //
    // Note: this initialization in conjunction with _conditionally_
    // calling materialize_bitmask() below covers the case
    // materialize_merged_bitmask_kernel<false, false>()
    // which won't be called anymore (because of the _condition_ below)
    //
    cudf::detail::set_null_mask(merged_view.null_mask(), 0, merged_view.size(), true, stream_);

    // set the null count:
    //
    merged_col->set_null_count(lcol.null_count() + rcol.null_count());

    using Type = device_storage_type_t<Element>;

    // to resolve view.data()'s types use: Element
    //
    auto const d_lcol = lcol.data<Type>();
    auto const d_rcol = rcol.data<Type>();

    auto exe_pol = rmm::exec_policy(stream_);

    // capture lcol, rcol
    // and "gather" into merged_view.data()[indx_merged]
    // from lcol or rcol, depending on side;
    //
    thrust::transform(exe_pol->on(stream_),
                      row_order_.begin(),
                      row_order_.end(),
                      merged_view.begin<Type>(),
                      [d_lcol, d_rcol] __device__(index_type const& index_pair) {
                        // When C++17, use structure bindings
                        auto side  = thrust::get<0>(index_pair);
                        auto index = thrust::get<1>(index_pair);
                        return side == side::LEFT ? d_lcol[index] : d_rcol[index];
                      });

    // CAVEAT: conditional call below is erroneous without
    // set_null_mask() call (see TODO above):
    //
    if (lcol.has_nulls() || rcol.has_nulls()) {
      // resolve null mask:
      //
      materialize_bitmask(lcol, rcol, merged_view, row_order_.data().get(), stream_);
    }

    return merged_col;
  }

 private:
  index_vector const& row_order_;
  rmm::mr::device_memory_resource* mr_;
  hipStream_t stream_;
};

// specialization for strings
template <>
std::unique_ptr<column> column_merger::operator()<cudf::string_view>(column_view const& lcol,
                                                                     column_view const& rcol) const
{
  auto column = strings::detail::merge<index_type>(strings_column_view(lcol),
                                                   strings_column_view(rcol),
                                                   row_order_.begin(),
                                                   row_order_.end(),
                                                   mr_,
                                                   stream_);
  if (lcol.has_nulls() || rcol.has_nulls()) {
    auto merged_view = column->mutable_view();
    materialize_bitmask(lcol, rcol, merged_view, row_order_.data().get(), stream_);
  }
  return column;
}

// specialization for dictionary
template <>
std::unique_ptr<column> column_merger::operator()<cudf::dictionary32>(column_view const& lcol,
                                                                      column_view const& rcol) const
{
  auto result = cudf::dictionary::detail::merge(cudf::dictionary_column_view(lcol),
                                                cudf::dictionary_column_view(rcol),
                                                row_order_,
                                                mr_,
                                                stream_);
  // set the validity mask
  if (lcol.has_nulls() || rcol.has_nulls()) {
    auto merged_view = result->mutable_view();
    materialize_bitmask(lcol, rcol, merged_view, row_order_.data().get(), stream_);
  }
  return result;
}

using table_ptr_type = std::unique_ptr<cudf::table>;

namespace {
table_ptr_type merge(cudf::table_view const& left_table,
                     cudf::table_view const& right_table,
                     std::vector<cudf::size_type> const& key_cols,
                     std::vector<cudf::order> const& column_order,
                     std::vector<cudf::null_order> const& null_precedence,
                     rmm::mr::device_memory_resource* mr,
                     hipStream_t stream = 0)
{
  // collect index columns for lhs, rhs, resp.
  //
  cudf::table_view index_left_view{left_table.select(key_cols)};
  cudf::table_view index_right_view{right_table.select(key_cols)};
  bool const nullable = cudf::has_nulls(index_left_view) || cudf::has_nulls(index_right_view);

  // extract merged row order according to indices:
  //
  rmm::device_vector<index_type> merged_indices = generate_merged_indices(
    index_left_view, index_right_view, column_order, null_precedence, nullable);

  // create merged table:
  //
  auto const n_cols = left_table.num_columns();
  std::vector<std::unique_ptr<column>> merged_cols;
  merged_cols.reserve(n_cols);

  column_merger merger{merged_indices, mr, stream};
  transform(left_table.begin(),
            left_table.end(),
            right_table.begin(),
            std::back_inserter(merged_cols),
            [&](auto const& left_col, auto const& right_col) {
              return cudf::type_dispatcher(left_col.type(), merger, left_col, right_col);
            });

  return std::make_unique<cudf::table>(std::move(merged_cols));
}

struct merge_queue_item {
  table_view view;
  table_ptr_type table;
  // Priority is a separate member to ensure that moving from an object
  // does not change its priority (which would ruin the queue invariant)
  cudf::size_type priority = 0;

  merge_queue_item(table_view const& view, table_ptr_type&& table)
    : view{view}, table{std::move(table)}, priority{-view.num_rows()}
  {
  }

  bool operator<(merge_queue_item const& other) const { return priority < other.priority; }
};

// Helper function to ensure that moving out of the priority_queue is "atomic"
template <typename T>
T top_and_pop(std::priority_queue<T>& q)
{
  auto moved = std::move(const_cast<T&>(q.top()));
  q.pop();
  return moved;
}

}  // namespace

table_ptr_type merge(std::vector<table_view> const& tables_to_merge,
                     std::vector<cudf::size_type> const& key_cols,
                     std::vector<cudf::order> const& column_order,
                     std::vector<cudf::null_order> const& null_precedence,
                     rmm::mr::device_memory_resource* mr,
                     hipStream_t stream = 0)
{
  if (tables_to_merge.empty()) { return std::make_unique<cudf::table>(); }

  auto const& first_table = tables_to_merge.front();
  auto const n_cols       = first_table.num_columns();

  CUDF_EXPECTS(std::all_of(tables_to_merge.cbegin(),
                           tables_to_merge.cend(),
                           [n_cols](auto const& tbl) { return n_cols == tbl.num_columns(); }),
               "Mismatched number of columns");
  CUDF_EXPECTS(
    std::all_of(tables_to_merge.cbegin(),
                tables_to_merge.cend(),
                [&](auto const& tbl) { return cudf::have_same_types(first_table, tbl); }),
    "Mismatched column types");

  CUDF_EXPECTS(!key_cols.empty(), "Empty key_cols");
  CUDF_EXPECTS(key_cols.size() <= static_cast<size_t>(n_cols), "Too many values in key_cols");

  CUDF_EXPECTS(key_cols.size() == column_order.size(),
               "Mismatched size between key_cols and column_order");

  // This utility will ensure all corresponding dictionary columns have matching keys.
  // It will return any new dictionary columns created as well as updated table_views.
  auto matched = cudf::dictionary::detail::match_dictionaries(
    tables_to_merge, rmm::mr::get_current_device_resource(), stream);
  auto merge_tables = matched.second;

  // A queue of (table view, table) pairs
  std::priority_queue<merge_queue_item> merge_queue;
  // The table pointer is null if we do not own the table (input tables)
  std::for_each(merge_tables.begin(), merge_tables.end(), [&](auto const& table) {
    if (table.num_rows() > 0) merge_queue.emplace(table, table_ptr_type());
  });

  // If there is only one non-empty table_view, return its copy
  if (merge_queue.size() == 1) { return std::make_unique<cudf::table>(merge_queue.top().view); }
  // No inputs have rows, return a table with same columns as the first one
  if (merge_queue.empty()) { return empty_like(first_table); }

  // Pick the two smallest tables and merge them
  // Until there is only one table left in the queue
  while (merge_queue.size() > 1) {
    // To delete the intermediate table at the end of the block
    auto const left_table = top_and_pop(merge_queue);
    // Deallocated at the end of the block
    auto const right_table = top_and_pop(merge_queue);

    // Only use mr for the output table
    auto const& new_tbl_rm = merge_queue.empty() ? mr : rmm::mr::get_current_device_resource();
    auto merged_table      = merge(left_table.view,
                              right_table.view,
                              key_cols,
                              column_order,
                              null_precedence,
                              new_tbl_rm,
                              stream);

    auto const merged_table_view = merged_table->view();
    merge_queue.emplace(merged_table_view, std::move(merged_table));
  }

  return std::move(top_and_pop(merge_queue).table);
}

}  // namespace detail

std::unique_ptr<cudf::table> merge(std::vector<table_view> const& tables_to_merge,
                                   std::vector<cudf::size_type> const& key_cols,
                                   std::vector<cudf::order> const& column_order,
                                   std::vector<cudf::null_order> const& null_precedence,
                                   rmm::mr::device_memory_resource* mr)
{
  CUDF_FUNC_RANGE();
  return detail::merge(tables_to_merge, key_cols, column_order, null_precedence, mr);
}

}  // namespace cudf
