#include "hip/hip_runtime.h"

#include <thrust/execution_policy.h>
#include <thrust/for_each.h>
#include <thrust/iterator/constant_iterator.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/tuple.h>
#include <thrust/device_vector.h>
#include <thrust/merge.h>
#include <algorithm>
#include <utility>
#include <vector>
#include <memory>
#include <nvstrings/NVCategory.h>

#include <cudf/cudf.h>
#include <cudf/types.hpp>
#include <cudf/copying.hpp>
#include <cudf/table.hpp>
#include "table/device_table.cuh"
#include "table/device_table_row_operators.cuh"
#include "bitmask/legacy/bit_mask.cuh"
#include "string/nvcategory_util.hpp"
#include "rmm/thrust_rmm_allocator.h"
#include "utilities/cuda_utils.hpp"

namespace {

/**
 * @brief Source table identifier to copy data from.
 */
enum class side : bool { LEFT, RIGHT };

using bit_mask::bit_mask_t;
using index_type = thrust::tuple<side, gdf_size_type>; // `thrust::get<0>` indicates left/right side, `thrust::get<1>` indicates the row index

/**
 * @brief Merges the bits of two validity bitmasks.
 *
 * Merges the bits from two source bitmask into the destination bitmask
 * according to `merged_indices` map such that bit `i` in `destination_mask`
 * will be equal to bit `thrust::get<1>(merged_indices[i])` from `source_left_mask`
 * if `thrust::get<0>(merged_indices[i])` equals `side::LEFT`; otherwise,
 * from `source_right_mask`.
 *
 * `source_left_mask`, `source_right_mask` and `destination_mask` must not
 * overlap.
 *
 * @tparam left_have_valids Indicates whether source_left_mask is null
 * @tparam right_have_valids Indicates whether source_right_mask is null
 * @param[in] source_left_mask The left mask whose bits will be merged
 * @param[in] source_right_mask The right mask whose bits will be merged
 * @param[out] destination_mask The output mask after merging the left and right masks
 * @param[in] num_destination_rows The number of bits in the destination_mask
 * @param[in] merged_indices The map that indicates from which input mask and which bit
 * will be copied to the output. Length must be equal to `num_destination_rows`
 */
template <bool left_have_valids, bool right_have_valids>
__global__ void materialize_merged_bitmask_kernel(
    bit_mask_t const* const __restrict__ source_left_mask,
    bit_mask_t const* const __restrict__ source_right_mask,
    bit_mask_t* const destination_mask,
    gdf_size_type const num_destination_rows,
    index_type const* const __restrict__ merged_indices) {

  gdf_index_type destination_row = threadIdx.x + blockIdx.x * blockDim.x;

  auto active_threads =
      __ballot_sync(0xffffffff, destination_row < num_destination_rows);

  while (destination_row < num_destination_rows) {
    index_type const& merged_idx = merged_indices[destination_row];
    side const src_side = thrust::get<0>(merged_idx);
    gdf_size_type const src_row  = thrust::get<1>(merged_idx);
    bool const from_left{src_side == side::LEFT};
    bool source_bit_is_valid{true};
    if (left_have_valids && from_left) {
        source_bit_is_valid = bit_mask::is_valid(source_left_mask, src_row);
    }
    else if (right_have_valids && !from_left) {
        source_bit_is_valid = bit_mask::is_valid(source_right_mask, src_row);
    }
    
    // Use ballot to find all valid bits in this warp and create the output
    // bitmask element
    bit_mask_t const result_mask{
        __ballot_sync(active_threads, source_bit_is_valid)};

    gdf_index_type const output_element = cudf::util::detail::bit_container_index<bit_mask_t, gdf_index_type>(destination_row);
    
    // Only one thread writes output
    if (0 == threadIdx.x % warpSize) {
      destination_mask[output_element] = result_mask;
    }

    destination_row += blockDim.x * gridDim.x;
    active_threads =
        __ballot_sync(active_threads, destination_row < num_destination_rows);
  }
}

void materialize_bitmask(gdf_column const* left_col,
                        gdf_column const* right_col,
                        gdf_column* out_col,
                        index_type const* merged_indices,
                        hipStream_t stream) {
    constexpr gdf_size_type BLOCK_SIZE{256};
    cudf::util::cuda::grid_config_1d grid_config {out_col->size, BLOCK_SIZE };

    bit_mask_t* left_valid = reinterpret_cast<bit_mask_t*>(left_col->valid);
    bit_mask_t* right_valid = reinterpret_cast<bit_mask_t*>(right_col->valid);
    bit_mask_t* out_valid = reinterpret_cast<bit_mask_t*>(out_col->valid);
    if (left_valid) {
        if (right_valid) {
            materialize_merged_bitmask_kernel<true, true>
            <<<grid_config.num_blocks, grid_config.num_threads_per_block, 0, stream>>>
            (left_valid, right_valid, out_valid, out_col->size, merged_indices);
        } else {
            materialize_merged_bitmask_kernel<true, false>
            <<<grid_config.num_blocks, grid_config.num_threads_per_block, 0, stream>>>
            (left_valid, right_valid, out_valid, out_col->size, merged_indices);
        }
    } else {
        if (right_valid) {
            materialize_merged_bitmask_kernel<false, true>
            <<<grid_config.num_blocks, grid_config.num_threads_per_block, 0, stream>>>
            (left_valid, right_valid, out_valid, out_col->size, merged_indices);
        } else {
            materialize_merged_bitmask_kernel<false, false>
            <<<grid_config.num_blocks, grid_config.num_threads_per_block, 0, stream>>>
            (left_valid, right_valid, out_valid, out_col->size, merged_indices);
        }
    }

    CHECK_STREAM(stream);
}

rmm::device_vector<index_type>
generate_merged_indices(device_table const& left_table,
                        device_table const& right_table,
                        rmm::device_vector<int8_t> const& asc_desc,
                        bool nulls_are_smallest,
                        hipStream_t stream) {

    const gdf_size_type left_size  = left_table.num_rows();
    const gdf_size_type right_size = right_table.num_rows();
    const gdf_size_type total_size = left_size + right_size;

    thrust::constant_iterator<side> left_side(side::LEFT);
    thrust::constant_iterator<side> right_side(side::RIGHT);

    auto left_indices = thrust::make_counting_iterator(static_cast<gdf_size_type>(0));
    auto right_indices = thrust::make_counting_iterator(static_cast<gdf_size_type>(0));

    auto left_begin_zip_iterator = thrust::make_zip_iterator(thrust::make_tuple(left_side, left_indices));
    auto right_begin_zip_iterator = thrust::make_zip_iterator(thrust::make_tuple(right_side, right_indices));

    auto left_end_zip_iterator = thrust::make_zip_iterator(thrust::make_tuple(left_side + left_size, left_indices + left_size));
    auto right_end_zip_iterator = thrust::make_zip_iterator(thrust::make_tuple(right_side + right_size, right_indices + right_size));

    rmm::device_vector<index_type> merged_indices(total_size);
    bool nullable = left_table.has_nulls() || right_table.has_nulls();
    if (nullable){
        auto ineq_op = row_inequality_comparator<true>(right_table, left_table, nulls_are_smallest, asc_desc.data().get()); 
        thrust::merge(rmm::exec_policy(stream)->on(stream),
                    left_begin_zip_iterator,
                    left_end_zip_iterator,
                    right_begin_zip_iterator,
                    right_end_zip_iterator,
                    merged_indices.begin(),
                    [=] __device__ (thrust::tuple<side, gdf_size_type> const & right_tuple,
                                    thrust::tuple<side, gdf_size_type> const & left_tuple) {
                        return ineq_op(thrust::get<1>(right_tuple), thrust::get<1>(left_tuple));
                    });			        
    } else {
        auto ineq_op = row_inequality_comparator<false>(right_table, left_table, nulls_are_smallest, asc_desc.data().get()); 
        thrust::merge(rmm::exec_policy(stream)->on(stream),
                    left_begin_zip_iterator,
                    left_end_zip_iterator,
                    right_begin_zip_iterator,
                    right_end_zip_iterator,
                    merged_indices.begin(),
                    [=] __device__ (thrust::tuple<side, gdf_size_type> const & right_tuple,
                                    thrust::tuple<side, gdf_size_type> const & left_tuple) {
                        return ineq_op(thrust::get<1>(right_tuple), thrust::get<1>(left_tuple));
                    });					        
    }

    CHECK_STREAM(stream);

    return merged_indices;
}

} // namespace

namespace cudf {
namespace detail {

table merge(table const& left_table,
            table const& right_table,
            std::vector<gdf_size_type> const& key_cols,
            std::vector<order_by_type> const& asc_desc,
            bool nulls_are_smallest,
            hipStream_t stream = 0) {
    CUDF_EXPECTS(left_table.num_columns() == right_table.num_columns(), "Mismatched number of columns");
    if (left_table.num_columns() == 0) {
        return cudf::empty_like(left_table);
    }
    
    std::vector<gdf_dtype> left_table_dtypes = cudf::column_dtypes(left_table);
    std::vector<gdf_dtype> right_table_dtypes = cudf::column_dtypes(right_table);
    CUDF_EXPECTS(std::equal(left_table_dtypes.cbegin(), left_table_dtypes.cend(), right_table_dtypes.cbegin(), right_table_dtypes.cend()), "Mismatched column dtypes");
    CUDF_EXPECTS(key_cols.size() > 0, "Empty key_cols");
    CUDF_EXPECTS(key_cols.size() <= static_cast<size_t>(left_table.num_columns()), "Too many values in key_cols");
    CUDF_EXPECTS(asc_desc.size() > 0, "Empty asc_desc");
    CUDF_EXPECTS(asc_desc.size() <= static_cast<size_t>(left_table.num_columns()), "Too many values in asc_desc");
    CUDF_EXPECTS(key_cols.size() == asc_desc.size(), "Mismatched size between key_cols and asc_desc");


    auto gdf_col_deleter = [](gdf_column *col) {
        gdf_column_free(col);
        delete col;
    };
    using gdf_col_ptr = typename std::unique_ptr<gdf_column, decltype(gdf_col_deleter)>;
    std::vector<gdf_col_ptr> temp_columns_to_free;
    std::vector<gdf_column*> left_cols_sync(const_cast<gdf_column**>(left_table.begin()), const_cast<gdf_column**>(left_table.end()));
    std::vector<gdf_column*> right_cols_sync(const_cast<gdf_column**>(right_table.begin()), const_cast<gdf_column**>(right_table.end()));
    for (gdf_size_type i = 0; i < left_table.num_columns(); i++) {
        gdf_column * left_col = const_cast<gdf_column*>(left_table.get_column(i));
        gdf_column * right_col = const_cast<gdf_column*>(right_table.get_column(i));
        
        if (left_col->dtype != GDF_STRING_CATEGORY){
            continue;
        }

        // If the inputs are nvcategory we need to make the dictionaries comparable

        temp_columns_to_free.push_back(gdf_col_ptr(new gdf_column{}, gdf_col_deleter));
        gdf_column * new_left_column_ptr = temp_columns_to_free.back().get();
        temp_columns_to_free.push_back(gdf_col_ptr(new gdf_column{}, gdf_col_deleter));
        gdf_column * new_right_column_ptr = temp_columns_to_free.back().get();

        *new_left_column_ptr = allocate_like(*left_col, true, stream);
        if (new_left_column_ptr->valid) {
            CUDA_TRY( hipMemcpyAsync(new_left_column_ptr->valid, left_col->valid, sizeof(gdf_valid_type)*gdf_num_bitmask_elements(left_col->size), hipMemcpyDefault, stream) );
            new_left_column_ptr->null_count = left_col->null_count;
        }
        
        *new_right_column_ptr = allocate_like(*right_col, true, stream);
        if (new_right_column_ptr->valid) {
            CUDA_TRY( hipMemcpyAsync(new_right_column_ptr->valid, right_col->valid, sizeof(gdf_valid_type)*gdf_num_bitmask_elements(right_col->size), hipMemcpyDefault, stream) );
            new_right_column_ptr->null_count = right_col->null_count;
        }

        gdf_column * tmp_arr_input[2] = {left_col, right_col};
        gdf_column * tmp_arr_output[2] = {new_left_column_ptr, new_right_column_ptr};
        CUDF_TRY( sync_column_categories(tmp_arr_input, tmp_arr_output, 2) );

        left_cols_sync[i] = new_left_column_ptr;
        right_cols_sync[i] = new_right_column_ptr;
    }

    table left_sync_table(left_cols_sync);
    table right_sync_table(right_cols_sync);

    std::vector<gdf_column*> left_key_cols_vect(key_cols.size());
    std::transform(key_cols.cbegin(), key_cols.cend(), left_key_cols_vect.begin(),
                  [&] (gdf_index_type const index) { return left_sync_table.get_column(index); });
    
    std::vector<gdf_column*> right_key_cols_vect(key_cols.size());
    std::transform(key_cols.cbegin(), key_cols.cend(), right_key_cols_vect.begin(),
                  [&] (gdf_index_type const index) { return right_sync_table.get_column(index); });

    auto left_key_table = device_table::create(left_key_cols_vect.size(), left_key_cols_vect.data());
    auto right_key_table = device_table::create(right_key_cols_vect.size(), right_key_cols_vect.data());
    rmm::device_vector<int8_t> asc_desc_d(asc_desc);

    rmm::device_vector<index_type> merged_indices = generate_merged_indices(*left_key_table, *right_key_table, asc_desc_d, nulls_are_smallest, stream);

    // Allocate output table
    bool nullable = has_nulls(left_sync_table) || has_nulls(right_sync_table);
    table destination_table(left_sync_table.num_rows() + right_sync_table.num_rows(), column_dtypes(left_sync_table), nullable, false, stream);
    for (gdf_size_type i = 0; i < destination_table.num_columns(); i++) {
        gdf_column const* left_col = left_sync_table.get_column(i);
        gdf_column * out_col = destination_table.get_column(i);
        
        if (left_col->dtype != GDF_STRING_CATEGORY){
            continue;
        }

        NVCategory * category = static_cast<NVCategory*>(left_col->dtype_info.category);
        out_col->dtype_info.category = category->copy();
    }
    
    // Materialize
    auto left_device_table_ptr = device_table::create(left_sync_table, stream);
    auto right_device_table_ptr = device_table::create(right_sync_table, stream);
    auto output_device_table_ptr = device_table::create(destination_table, stream);
    auto& left_device_table = *left_device_table_ptr;
    auto& right_device_table = *right_device_table_ptr;
    auto& output_device_table = *output_device_table_ptr;

    auto index_start_it = thrust::make_zip_iterator(thrust::make_tuple(
                                                    thrust::make_counting_iterator(static_cast<gdf_size_type>(0)), 
                                                    merged_indices.begin()));
    auto index_end_it = thrust::make_zip_iterator(thrust::make_tuple(
                                                thrust::make_counting_iterator(static_cast<gdf_size_type>(merged_indices.size())),
                                                merged_indices.end()));

    thrust::for_each(rmm::exec_policy(stream)->on(stream),
                    index_start_it,
                    index_end_it,
                    [=] __device__ (auto const & idx_tuple){
                        gdf_size_type dest_row = thrust::get<0>(idx_tuple);
                        index_type merged_idx = thrust::get<1>(idx_tuple);
                        side src_side = thrust::get<0>(merged_idx);
                        gdf_size_type src_row  = thrust::get<1>(merged_idx);
                        device_table const & src_device_table = src_side == side::LEFT ? left_device_table : right_device_table;
                        copy_row<false>(output_device_table, dest_row, src_device_table, src_row);
                    });
    
    CHECK_STREAM(0);

    if (nullable) {
        for (gdf_size_type i = 0; i < destination_table.num_columns(); i++) {
            gdf_column const* left_col = left_sync_table.get_column(i);
            gdf_column const* right_col = right_sync_table.get_column(i);
            gdf_column* out_col = destination_table.get_column(i);
            
            materialize_bitmask(left_col, right_col, out_col, merged_indices.data().get(), stream);
            
            out_col->null_count = left_col->null_count + right_col->null_count;
        }
    }

    return destination_table;
}

}  // namespace detail

table merge(table const& left_table,
            table const& right_table,
            std::vector<gdf_size_type> const& key_cols,
            std::vector<order_by_type> const& asc_desc,
            bool nulls_are_smallest) {
    return detail::merge(left_table, right_table, key_cols, asc_desc, nulls_are_smallest);
}

}  // namespace cudf
