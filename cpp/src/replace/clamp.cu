/*
 * Copyright (c) 2019-2020, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <cudf/column/column.hpp>
#include <cudf/column/column_device_view.cuh>
#include <cudf/column/column_factories.hpp>
#include <cudf/detail/copy.hpp>
#include <cudf/detail/iterator.cuh>
#include <cudf/detail/nvtx/ranges.hpp>
#include <cudf/dictionary/detail/search.hpp>
#include <cudf/dictionary/detail/update_keys.hpp>
#include <cudf/dictionary/dictionary_column_view.hpp>
#include <cudf/dictionary/dictionary_factories.hpp>
#include <cudf/replace.hpp>
#include <cudf/scalar/scalar.hpp>
#include <cudf/scalar/scalar_device_view.cuh>
#include <cudf/strings/detail/utilities.cuh>
#include <cudf/strings/detail/utilities.hpp>
#include <cudf/strings/strings_column_view.hpp>
#include <cudf/types.hpp>

namespace cudf {
namespace detail {
namespace {
template <typename Transformer>
std::pair<std::unique_ptr<column>, std::unique_ptr<column>> form_offsets_and_char_column(
  cudf::column_device_view input,
  size_type null_count,
  Transformer offsets_transformer,
  rmm::mr::device_memory_resource* mr,
  hipStream_t stream)
{
  std::unique_ptr<column> offsets_column{};
  auto strings_count = input.size();

  if (input.nullable()) {
    auto input_begin =
      cudf::detail::make_null_replacement_iterator<string_view>(input, string_view{});
    auto offsets_transformer_itr =
      thrust::make_transform_iterator(input_begin, offsets_transformer);
    offsets_column = cudf::strings::detail::make_offsets_child_column(
      offsets_transformer_itr, offsets_transformer_itr + strings_count, mr, stream);
  } else {
    auto offsets_transformer_itr =
      thrust::make_transform_iterator(input.begin<string_view>(), offsets_transformer);
    offsets_column = cudf::strings::detail::make_offsets_child_column(
      offsets_transformer_itr, offsets_transformer_itr + strings_count, mr, stream);
  }

  auto d_offsets = offsets_column->view().template data<size_type>();
  // build chars column
  size_type bytes = thrust::device_pointer_cast(d_offsets)[strings_count];
  auto chars_column =
    cudf::strings::detail::create_chars_child_column(strings_count, null_count, bytes, mr, stream);

  return std::make_pair(std::move(offsets_column), std::move(chars_column));
}

template <typename ScalarIterator>
std::unique_ptr<cudf::column> clamp_string_column(strings_column_view const& input,
                                                  ScalarIterator const& lo_itr,
                                                  ScalarIterator const& lo_replace_itr,
                                                  ScalarIterator const& hi_itr,
                                                  ScalarIterator const& hi_replace_itr,
                                                  rmm::mr::device_memory_resource* mr,
                                                  hipStream_t stream)
{
  auto input_device_column = column_device_view::create(input.parent(), stream);
  auto d_input             = *input_device_column;
  size_type null_count     = input.parent().null_count();

  // build offset column
  auto offsets_transformer = [lo_itr, hi_itr, lo_replace_itr, hi_replace_itr] __device__(
                               string_view element, bool is_valid = true) {
    const auto d_lo         = (*lo_itr).first;
    const auto d_hi         = (*hi_itr).first;
    const auto d_lo_replace = (*lo_replace_itr).first;
    const auto d_hi_replace = (*hi_replace_itr).first;
    const auto lo_valid     = (*lo_itr).second;
    const auto hi_valid     = (*hi_itr).second;
    size_type bytes         = 0;

    if (is_valid) {
      if (lo_valid and element < d_lo) {
        bytes = d_lo_replace.size_bytes();
      } else if (hi_valid and d_hi < element) {
        bytes = d_hi_replace.size_bytes();
      } else {
        bytes = element.size_bytes();
      }
    }
    return bytes;
  };

  auto offset_and_char =
    form_offsets_and_char_column(d_input, null_count, offsets_transformer, mr, stream);
  auto offsets_column(std::move(offset_and_char.first));
  auto chars_column(std::move(offset_and_char.second));

  auto d_offsets = offsets_column->view().template data<size_type>();
  auto d_chars   = chars_column->mutable_view().template data<char>();
  // fill in chars
  auto copy_transformer =
    [d_input, lo_itr, hi_itr, lo_replace_itr, hi_replace_itr, d_offsets, d_chars] __device__(
      size_type idx) {
      if (d_input.is_null(idx)) { return; }
      auto input_element      = d_input.element<string_view>(idx);
      const auto d_lo         = (*lo_itr).first;
      const auto d_hi         = (*hi_itr).first;
      const auto d_lo_replace = (*lo_replace_itr).first;
      const auto d_hi_replace = (*hi_replace_itr).first;
      const auto lo_valid     = (*lo_itr).second;
      const auto hi_valid     = (*hi_itr).second;

      if (lo_valid and input_element < d_lo) {
        memcpy(d_chars + d_offsets[idx], d_lo_replace.data(), d_lo_replace.size_bytes());
      } else if (hi_valid and d_hi < input_element) {
        memcpy(d_chars + d_offsets[idx], d_hi_replace.data(), d_hi_replace.size_bytes());
      } else {
        memcpy(d_chars + d_offsets[idx], input_element.data(), input_element.size_bytes());
      }
    };

  auto exec = rmm::exec_policy(stream);
  thrust::for_each_n(
    exec->on(stream), thrust::make_counting_iterator<size_type>(0), input.size(), copy_transformer);

  return make_strings_column(input.size(),
                             std::move(offsets_column),
                             std::move(chars_column),
                             input.null_count(),
                             std::move(copy_bitmask(input.parent())),
                             stream,
                             mr);
}

template <typename T, typename ScalarIterator>
std::enable_if_t<cudf::is_fixed_width<T>(), std::unique_ptr<cudf::column>> clamper(
  column_view const& input,
  ScalarIterator const& lo_itr,
  ScalarIterator const& lo_replace_itr,
  ScalarIterator const& hi_itr,
  ScalarIterator const& hi_replace_itr,
  rmm::mr::device_memory_resource* mr,
  hipStream_t stream)
{
  auto output =
    detail::allocate_like(input, input.size(), mask_allocation_policy::NEVER, stream, mr);
  // mask will not change
  if (input.nullable()) { output->set_null_mask(copy_bitmask(input), input.null_count()); }

  auto output_device_view =
    cudf::mutable_column_device_view::create(output->mutable_view(), stream);
  auto input_device_view = cudf::column_device_view::create(input, stream);
  auto scalar_zip_itr =
    thrust::make_zip_iterator(thrust::make_tuple(lo_itr, lo_replace_itr, hi_itr, hi_replace_itr));

  auto trans = [] __device__(auto element_validity_pair, auto scalar_tuple) {
    if (element_validity_pair.second) {
      auto lo_validity_pair = thrust::get<0>(scalar_tuple);
      auto hi_validity_pair = thrust::get<2>(scalar_tuple);
      if (lo_validity_pair.second and (element_validity_pair.first < lo_validity_pair.first)) {
        return thrust::get<1>(scalar_tuple).first;
      } else if (hi_validity_pair.second and
                 (element_validity_pair.first > hi_validity_pair.first)) {
        return thrust::get<3>(scalar_tuple).first;
      }
    }

    return element_validity_pair.first;
  };

  if (input.has_nulls()) {
    auto input_pair_iterator = make_pair_iterator<T, true>(*input_device_view);
    thrust::transform(rmm::exec_policy(stream)->on(stream),
                      input_pair_iterator,
                      input_pair_iterator + input.size(),
                      scalar_zip_itr,
                      output_device_view->begin<T>(),
                      trans);
  } else {
    auto input_pair_iterator = make_pair_iterator<T, false>(*input_device_view);
    thrust::transform(rmm::exec_policy(stream)->on(stream),
                      input_pair_iterator,
                      input_pair_iterator + input.size(),
                      scalar_zip_itr,
                      output_device_view->begin<T>(),
                      trans);
  }

  return output;
}

template <typename T, typename ScalarIterator>
std::enable_if_t<std::is_same<T, string_view>::value, std::unique_ptr<cudf::column>> clamper(
  column_view const& input,
  ScalarIterator const& lo_itr,
  ScalarIterator const& lo_replace_itr,
  ScalarIterator const& hi_itr,
  ScalarIterator const& hi_replace_itr,
  rmm::mr::device_memory_resource* mr,
  hipStream_t stream)
{
  return clamp_string_column(input, lo_itr, lo_replace_itr, hi_itr, hi_replace_itr, mr, stream);
}

}  // namespace

template <typename T, typename ScalarIterator>
std::unique_ptr<column> clamp(
  column_view const& input,
  ScalarIterator const& lo_itr,
  ScalarIterator const& lo_replace_itr,
  ScalarIterator const& hi_itr,
  ScalarIterator const& hi_replace_itr,
  rmm::mr::device_memory_resource* mr = rmm::mr::get_current_device_resource(),
  hipStream_t stream                 = 0)
{
  return clamper<T>(input, lo_itr, lo_replace_itr, hi_itr, hi_replace_itr, mr, stream);
}

struct dispatch_clamp {
  template <typename T>
  std::unique_ptr<column> operator()(
    column_view const& input,
    scalar const& lo,
    scalar const& lo_replace,
    scalar const& hi,
    scalar const& hi_replace,
    rmm::mr::device_memory_resource* mr = rmm::mr::get_current_device_resource(),
    hipStream_t stream                 = 0)
  {
    CUDF_EXPECTS(lo.type() == input.type(), "mismatching types of scalar and input");

    using Type = device_storage_type_t<T>;

    auto lo_itr         = make_pair_iterator<Type>(lo);
    auto hi_itr         = make_pair_iterator<Type>(hi);
    auto lo_replace_itr = make_pair_iterator<Type>(lo_replace);
    auto hi_replace_itr = make_pair_iterator<Type>(hi_replace);

    return clamp<Type>(input, lo_itr, lo_replace_itr, hi_itr, hi_replace_itr, mr, stream);
  }
};

template <>
std::unique_ptr<column> dispatch_clamp::operator()<cudf::list_view>(
  column_view const& input,
  scalar const& lo,
  scalar const& lo_replace,
  scalar const& hi,
  scalar const& hi_replace,
  rmm::mr::device_memory_resource* mr,
  hipStream_t stream)
{
  CUDF_FAIL("clamp for list_view not supported");
}

template <>
std::unique_ptr<column> dispatch_clamp::operator()<struct_view>(column_view const& input,
                                                                scalar const& lo,
                                                                scalar const& lo_replace,
                                                                scalar const& hi,
                                                                scalar const& hi_replace,
                                                                rmm::mr::device_memory_resource* mr,
                                                                hipStream_t stream)
{
  CUDF_FAIL("clamp for struct_view not supported");
}

template <>
std::unique_ptr<column> dispatch_clamp::operator()<cudf::dictionary32>(
  column_view const& input,
  scalar const& lo,
  scalar const& lo_replace,
  scalar const& hi,
  scalar const& hi_replace,
  rmm::mr::device_memory_resource* mr,
  hipStream_t stream)
{
  // add lo_replace and hi_replace to keys
  auto matched_column = [&] {
    auto matched_view              = dictionary_column_view(input);
    std::unique_ptr<column> result = nullptr;
    auto add_scalar_key            = [&](scalar const& key, scalar const& key_replace) {
      if (key.is_valid()) {
        result = dictionary::detail::add_keys(
          matched_view, make_column_from_scalar(key_replace, 1, stream)->view(), mr, stream);
        matched_view = dictionary_column_view(result->view());
      }
    };
    add_scalar_key(lo, lo_replace);
    add_scalar_key(hi, hi_replace);
    return result;
  }();
  auto matched_view = dictionary_column_view(matched_column->view());

  // get the indexes for lo_replace and for hi_replace
  auto lo_replace_index = dictionary::detail::get_index(matched_view, lo_replace, stream);
  auto hi_replace_index = dictionary::detail::get_index(matched_view, hi_replace, stream);

  // get the closest indexes for lo and for hi
  auto lo_index = dictionary::detail::get_insert_index(matched_view, lo, stream);
  auto hi_index = dictionary::detail::get_insert_index(matched_view, hi, stream);

  // call clamp with the scalar indexes and the matched indices
  auto matched_indices = matched_view.get_indices_annotated();
  auto new_indices     = cudf::type_dispatcher(matched_indices.type(),
                                           dispatch_clamp{},
                                           matched_indices,
                                           *lo_index,
                                           *lo_replace_index,
                                           *hi_index,
                                           *hi_replace_index,
                                           mr,
                                           stream);

  auto const indices_type = new_indices->type();
  auto const output_size  = new_indices->size();
  auto const null_count   = new_indices->null_count();
  auto contents           = new_indices->release();
  auto indices_column     = std::make_unique<column>(indices_type,
                                                 static_cast<size_type>(output_size),
                                                 *(contents.data.release()),
                                                 rmm::device_buffer{0, stream, mr},
                                                 0);

  // take the keys from the matched column allocated using mr
  std::unique_ptr<column> keys_column(std::move(matched_column->release().children.back()));

  // create column with keys_column and indices_column
  return make_dictionary_column(std::move(keys_column),
                                std::move(indices_column),
                                std::move(*(contents.null_mask.release())),
                                null_count);
}

/**
 * @copydoc cudf::clamp(column_view const& input,
                                      scalar const& lo,
                                      scalar const& lo_replace,
                                      scalar const& hi,
                                      scalar const& hi_replace,
                                      rmm::mr::device_memory_resource* mr);
 *
 * @param[in] stream CUDA stream used for device memory operations and kernel launches.
 */
std::unique_ptr<column> clamp(
  column_view const& input,
  scalar const& lo,
  scalar const& lo_replace,
  scalar const& hi,
  scalar const& hi_replace,
  rmm::mr::device_memory_resource* mr = rmm::mr::get_current_device_resource(),
  hipStream_t stream                 = 0)
{
  CUDF_EXPECTS(lo.type() == hi.type(), "mismatching types of limit scalars");
  CUDF_EXPECTS(lo_replace.type() == hi_replace.type(), "mismatching types of replace scalars");
  CUDF_EXPECTS(lo.type() == lo_replace.type(), "mismatching types of limit and replace scalars");

  if ((not lo.is_valid(stream) and not hi.is_valid(stream)) or (input.is_empty())) {
    // There will be no change
    return std::make_unique<column>(input, stream, mr);
  }

  if (lo.is_valid(stream)) {
    CUDF_EXPECTS(lo_replace.is_valid(stream), "lo_replace can't be null if lo is not null");
  }
  if (hi.is_valid(stream)) {
    CUDF_EXPECTS(hi_replace.is_valid(stream), "hi_replace can't be null if hi is not null");
  }

  return cudf::type_dispatcher(
    input.type(), dispatch_clamp{}, input, lo, lo_replace, hi, hi_replace, mr, stream);
}

}  // namespace detail

// clamp input at lo and hi with lo_replace and hi_replace
std::unique_ptr<column> clamp(column_view const& input,
                              scalar const& lo,
                              scalar const& lo_replace,
                              scalar const& hi,
                              scalar const& hi_replace,
                              rmm::mr::device_memory_resource* mr)
{
  CUDF_FUNC_RANGE();
  return detail::clamp(input, lo, lo_replace, hi, hi_replace, mr);
}

// clamp input at lo and hi
std::unique_ptr<column> clamp(column_view const& input,
                              scalar const& lo,
                              scalar const& hi,
                              rmm::mr::device_memory_resource* mr)
{
  CUDF_FUNC_RANGE();
  return detail::clamp(input, lo, lo, hi, hi, mr);
}
}  // namespace cudf
