/*
 * Copyright (c) 2019-2020, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <cudf/column/column.hpp>
#include <cudf/column/column_device_view.cuh>
#include <cudf/column/column_factories.hpp>
#include <cudf/detail/copy.hpp>
#include <cudf/detail/iterator.cuh>
#include <cudf/detail/nvtx/ranges.hpp>
#include <cudf/replace.hpp>
#include <cudf/scalar/scalar.hpp>
#include <cudf/scalar/scalar_device_view.cuh>
#include <cudf/strings/detail/utilities.cuh>
#include <cudf/strings/detail/utilities.hpp>
#include <cudf/strings/strings_column_view.hpp>
#include <cudf/types.hpp>

namespace cudf {
namespace detail {
namespace {
template <typename Transformer>
std::pair<std::unique_ptr<column>, std::unique_ptr<column>> form_offsets_and_char_column(
  cudf::column_device_view input,
  size_type null_count,
  Transformer offsets_transformer,
  rmm::mr::device_memory_resource* mr,
  hipStream_t stream)
{
  std::unique_ptr<column> offsets_column{};
  auto strings_count = input.size();

  if (input.nullable()) {
    auto input_begin =
      cudf::detail::make_null_replacement_iterator<string_view>(input, string_view{});
    auto offsets_transformer_itr =
      thrust::make_transform_iterator(input_begin, offsets_transformer);
    offsets_column = cudf::strings::detail::make_offsets_child_column(
      offsets_transformer_itr, offsets_transformer_itr + strings_count, mr, stream);
  } else {
    auto offsets_transformer_itr =
      thrust::make_transform_iterator(input.begin<string_view>(), offsets_transformer);
    offsets_column = cudf::strings::detail::make_offsets_child_column(
      offsets_transformer_itr, offsets_transformer_itr + strings_count, mr, stream);
  }

  auto d_offsets = offsets_column->view().template data<size_type>();
  // build chars column
  size_type bytes = thrust::device_pointer_cast(d_offsets)[strings_count];
  auto chars_column =
    cudf::strings::detail::create_chars_child_column(strings_count, null_count, bytes, mr, stream);

  return std::make_pair(std::move(offsets_column), std::move(chars_column));
}

template <typename ScalarIterator>
std::unique_ptr<cudf::column> clamp_string_column(strings_column_view const& input,
                                                  ScalarIterator const& lo_itr,
                                                  ScalarIterator const& lo_replace_itr,
                                                  ScalarIterator const& hi_itr,
                                                  ScalarIterator const& hi_replace_itr,
                                                  rmm::mr::device_memory_resource* mr,
                                                  hipStream_t stream)
{
  auto input_device_column = column_device_view::create(input.parent(), stream);
  auto d_input             = *input_device_column;
  size_type null_count     = input.parent().null_count();

  // build offset column
  auto offsets_transformer = [lo_itr, hi_itr, lo_replace_itr, hi_replace_itr] __device__(
                               string_view element, bool is_valid = true) {
    const auto d_lo         = (*lo_itr).first;
    const auto d_hi         = (*hi_itr).first;
    const auto d_lo_replace = (*lo_replace_itr).first;
    const auto d_hi_replace = (*hi_replace_itr).first;
    const auto lo_valid     = (*lo_itr).second;
    const auto hi_valid     = (*hi_itr).second;
    size_type bytes         = 0;

    if (is_valid) {
      if (lo_valid and element < d_lo) {
        bytes = d_lo_replace.size_bytes();
      } else if (hi_valid and d_hi < element) {
        bytes = d_hi_replace.size_bytes();
      } else {
        bytes = element.size_bytes();
      }
    }
    return bytes;
  };

  auto offset_and_char =
    form_offsets_and_char_column(d_input, null_count, offsets_transformer, mr, stream);
  auto offsets_column(std::move(offset_and_char.first));
  auto chars_column(std::move(offset_and_char.second));

  auto d_offsets = offsets_column->view().template data<size_type>();
  auto d_chars   = chars_column->mutable_view().template data<char>();
  // fill in chars
  auto copy_transformer =
    [d_input, lo_itr, hi_itr, lo_replace_itr, hi_replace_itr, d_offsets, d_chars] __device__(
      size_type idx) {
      if (d_input.is_null(idx)) { return; }
      auto input_element      = d_input.element<string_view>(idx);
      const auto d_lo         = (*lo_itr).first;
      const auto d_hi         = (*hi_itr).first;
      const auto d_lo_replace = (*lo_replace_itr).first;
      const auto d_hi_replace = (*hi_replace_itr).first;
      const auto lo_valid     = (*lo_itr).second;
      const auto hi_valid     = (*hi_itr).second;

      if (lo_valid and input_element < d_lo) {
        memcpy(d_chars + d_offsets[idx], d_lo_replace.data(), d_lo_replace.size_bytes());
      } else if (hi_valid and d_hi < input_element) {
        memcpy(d_chars + d_offsets[idx], d_hi_replace.data(), d_hi_replace.size_bytes());
      } else {
        memcpy(d_chars + d_offsets[idx], input_element.data(), input_element.size_bytes());
      }
    };

  auto exec = rmm::exec_policy(stream);
  thrust::for_each_n(
    exec->on(stream), thrust::make_counting_iterator<size_type>(0), input.size(), copy_transformer);

  return make_strings_column(input.size(),
                             std::move(offsets_column),
                             std::move(chars_column),
                             input.null_count(),
                             std::move(copy_bitmask(input.parent())),
                             stream,
                             mr);
}

template <typename T, typename ScalarIterator>
std::enable_if_t<cudf::is_fixed_width<T>(), std::unique_ptr<cudf::column>> clamper(
  column_view const& input,
  ScalarIterator const& lo_itr,
  ScalarIterator const& lo_replace_itr,
  ScalarIterator const& hi_itr,
  ScalarIterator const& hi_replace_itr,
  rmm::mr::device_memory_resource* mr,
  hipStream_t stream)
{
  auto output =
    detail::allocate_like(input, input.size(), mask_allocation_policy::NEVER, mr, stream);
  // mask will not change
  if (input.nullable()) { output->set_null_mask(copy_bitmask(input), input.null_count()); }

  auto output_device_view =
    cudf::mutable_column_device_view::create(output->mutable_view(), stream);
  auto input_device_view = cudf::column_device_view::create(input, stream);
  auto scalar_zip_itr =
    thrust::make_zip_iterator(thrust::make_tuple(lo_itr, lo_replace_itr, hi_itr, hi_replace_itr));

  auto trans = [] __device__(auto element_validity_pair, auto scalar_tuple) {
    if (element_validity_pair.second) {
      auto lo_validity_pair = thrust::get<0>(scalar_tuple);
      auto hi_validity_pair = thrust::get<2>(scalar_tuple);
      if (lo_validity_pair.second and (element_validity_pair.first < lo_validity_pair.first)) {
        return thrust::get<1>(scalar_tuple).first;
      } else if (hi_validity_pair.second and
                 (element_validity_pair.first > hi_validity_pair.first)) {
        return thrust::get<3>(scalar_tuple).first;
      }
    }

    return element_validity_pair.first;
  };

  if (input.has_nulls()) {
    auto input_pair_iterator = make_pair_iterator<T, true>(*input_device_view);
    thrust::transform(rmm::exec_policy(stream)->on(stream),
                      input_pair_iterator,
                      input_pair_iterator + input.size(),
                      scalar_zip_itr,
                      output_device_view->begin<T>(),
                      trans);
  } else {
    auto input_pair_iterator = make_pair_iterator<T, false>(*input_device_view);
    thrust::transform(rmm::exec_policy(stream)->on(stream),
                      input_pair_iterator,
                      input_pair_iterator + input.size(),
                      scalar_zip_itr,
                      output_device_view->begin<T>(),
                      trans);
  }

  return output;
}

template <typename T, typename ScalarIterator>
std::enable_if_t<std::is_same<T, string_view>::value, std::unique_ptr<cudf::column>> clamper(
  column_view const& input,
  ScalarIterator const& lo_itr,
  ScalarIterator const& lo_replace_itr,
  ScalarIterator const& hi_itr,
  ScalarIterator const& hi_replace_itr,
  rmm::mr::device_memory_resource* mr,
  hipStream_t stream)
{
  return clamp_string_column(input, lo_itr, lo_replace_itr, hi_itr, hi_replace_itr, mr, stream);
}

template <typename T, typename ScalarIterator>
std::enable_if_t<std::is_same<T, dictionary32>::value, std::unique_ptr<cudf::column>> clamper(
  column_view const& input,
  ScalarIterator const& lo_itr,
  ScalarIterator const& lo_replace_itr,
  ScalarIterator const& hi_itr,
  ScalarIterator const& hi_replace_itr,
  rmm::mr::device_memory_resource* mr,
  hipStream_t stream)
{
  CUDF_FAIL("dictionary type not supported");
}

template <typename T, typename ScalarIterator>
std::enable_if_t<std::is_same<T, list_view>::value, std::unique_ptr<cudf::column>> clamper(
  column_view const& input,
  ScalarIterator const& lo_itr,
  ScalarIterator const& lo_replace_itr,
  ScalarIterator const& hi_itr,
  ScalarIterator const& hi_replace_itr,
  rmm::mr::device_memory_resource* mr,
  hipStream_t stream)
{
  CUDF_FAIL("list_view type not supported");
}

template <typename T, typename ScalarIterator>
std::enable_if_t<std::is_same<T, cudf::struct_view>::value, std::unique_ptr<cudf::column>> clamper(
  column_view const& input,
  ScalarIterator const& lo_itr,
  ScalarIterator const& lo_replace_itr,
  ScalarIterator const& hi_itr,
  ScalarIterator const& hi_replace_itr,
  rmm::mr::device_memory_resource* mr,
  hipStream_t stream)
{
  CUDF_FAIL("struct_view type not supported");
}

}  // namespace

template <typename T, typename ScalarIterator>
std::unique_ptr<column> clamp(
  column_view const& input,
  ScalarIterator const& lo_itr,
  ScalarIterator const& lo_replace_itr,
  ScalarIterator const& hi_itr,
  ScalarIterator const& hi_replace_itr,
  rmm::mr::device_memory_resource* mr = rmm::mr::get_current_device_resource(),
  hipStream_t stream                 = 0)
{
  return clamper<T>(input, lo_itr, lo_replace_itr, hi_itr, hi_replace_itr, mr, stream);
}

struct dispatch_clamp {
  template <typename T>
  std::unique_ptr<column> operator()(
    column_view const& input,
    scalar const& lo,
    scalar const& lo_replace,
    scalar const& hi,
    scalar const& hi_replace,
    rmm::mr::device_memory_resource* mr = rmm::mr::get_current_device_resource(),
    hipStream_t stream                 = 0)
  {
    auto lo_itr         = make_pair_iterator<T>(lo);
    auto hi_itr         = make_pair_iterator<T>(hi);
    auto lo_replace_itr = make_pair_iterator<T>(lo_replace);
    auto hi_replace_itr = make_pair_iterator<T>(hi_replace);

    return clamp<T>(input, lo_itr, lo_replace_itr, hi_itr, hi_replace_itr, mr, stream);
  }
};

template <>
std::unique_ptr<column> dispatch_clamp::operator()<cudf::list_view>(
  column_view const& input,
  scalar const& lo,
  scalar const& lo_replace,
  scalar const& hi,
  scalar const& hi_replace,
  rmm::mr::device_memory_resource* mr,
  hipStream_t stream)
{
  CUDF_FAIL("clamp for list_view not supported");
}

template <>
std::unique_ptr<column> dispatch_clamp::operator()<numeric::decimal32>(
  column_view const& input,
  scalar const& lo,
  scalar const& lo_replace,
  scalar const& hi,
  scalar const& hi_replace,
  rmm::mr::device_memory_resource* mr,
  hipStream_t stream)
{
  CUDF_FAIL("clamp for decimal32 not supported");
}

template <>
std::unique_ptr<column> dispatch_clamp::operator()<numeric::decimal64>(
  column_view const& input,
  scalar const& lo,
  scalar const& lo_replace,
  scalar const& hi,
  scalar const& hi_replace,
  rmm::mr::device_memory_resource* mr,
  hipStream_t stream)
{
  CUDF_FAIL("clamp for decimal32 not supported");
}

template <>
std::unique_ptr<column> dispatch_clamp::operator()<struct_view>(column_view const& input,
                                                                scalar const& lo,
                                                                scalar const& lo_replace,
                                                                scalar const& hi,
                                                                scalar const& hi_replace,
                                                                rmm::mr::device_memory_resource* mr,
                                                                hipStream_t stream)
{
  CUDF_FAIL("clamp for struct_view not supported");
}

/**
 * @copydoc cudf::clamp(column_view const& input,
                                      scalar const& lo,
                                      scalar const& lo_replace,
                                      scalar const& hi,
                                      scalar const& hi_replace,
                                      rmm::mr::device_memory_resource* mr);
 *
 * @param[in] stream CUDA stream used for device memory operations and kernel launches.
 */
std::unique_ptr<column> clamp(
  column_view const& input,
  scalar const& lo,
  scalar const& lo_replace,
  scalar const& hi,
  scalar const& hi_replace,
  rmm::mr::device_memory_resource* mr = rmm::mr::get_current_device_resource(),
  hipStream_t stream                 = 0)
{
  CUDF_EXPECTS(lo.type() == hi.type(), "mismatching types of limit scalars");
  CUDF_EXPECTS(lo_replace.type() == hi_replace.type(), "mismatching types of replace scalars");
  CUDF_EXPECTS(lo.type() == lo_replace.type(), "mismatching types of limit and replace scalars");
  CUDF_EXPECTS(lo.type() == input.type(), "mismatching types of scalar and input");

  if ((not lo.is_valid(stream) and not hi.is_valid(stream)) or (input.is_empty())) {
    // There will be no change
    return std::make_unique<column>(input, stream, mr);
  }

  if (lo.is_valid(stream)) {
    CUDF_EXPECTS(lo_replace.is_valid(stream), "lo_replace can't be null if lo is not null");
  }
  if (hi.is_valid(stream)) {
    CUDF_EXPECTS(hi_replace.is_valid(stream), "hi_replace can't be null if hi is not null");
  }

  return cudf::type_dispatcher(
    input.type(), dispatch_clamp{}, input, lo, lo_replace, hi, hi_replace, mr, stream);
}

}  // namespace detail

// clamp input at lo and hi with lo_replace and hi_replace
std::unique_ptr<column> clamp(column_view const& input,
                              scalar const& lo,
                              scalar const& lo_replace,
                              scalar const& hi,
                              scalar const& hi_replace,
                              rmm::mr::device_memory_resource* mr)
{
  CUDF_FUNC_RANGE();
  return detail::clamp(input, lo, lo_replace, hi, hi_replace, mr);
}

// clamp input at lo and hi
std::unique_ptr<column> clamp(column_view const& input,
                              scalar const& lo,
                              scalar const& hi,
                              rmm::mr::device_memory_resource* mr)
{
  CUDF_FUNC_RANGE();
  return detail::clamp(input, lo, lo, hi, hi, mr);
}
}  // namespace cudf
