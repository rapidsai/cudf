#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2018-2019, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <thrust/device_ptr.h>
#include <thrust/find.h>
#include <thrust/execution_policy.h>

#include <cudf/cudf.h>
#include <rmm/rmm.h>
#include <cudf/types.hpp>
#include <cudf/utilities/error.hpp>
#include <cudf/utilities/legacy/type_dispatcher.hpp>
#include <cudf/column/column_device_view.cuh>
#include <cudf/column/column_factories.hpp>
#include <cudf/utilities/type_dispatcher.hpp>
#include <utilities/cudf_utils.h>
#include <utilities/cuda_utils.hpp>
#include <utilities/column_utils.hpp>

namespace {  // anonymous

using namespace cudf;       

template <typename T>
struct normalize_nans_and_zeros_lambda {
   column_device_view in;
   T __device__ operator()(size_type i)
   {
      auto e = in.element<T>(i);
      if (isnan(e)) {
         return std::numeric_limits<T>::quiet_NaN();
      }
      if (T{0.0} == e) {
         return T{0.0};
      }
      return e;
   }
};
   
/* --------------------------------------------------------------------------*/
/**
* @brief Functor called by the `type_dispatcher` in order to invoke and instantiate
*        `normalize_nans_and_zeros` with the appropriate data types.
*/
/* ----------------------------------------------------------------------------*/
struct normalize_nans_and_zeros_kernel_forwarder {
   // floats and doubles. what we really care about.
   template <typename T, std::enable_if_t<std::is_floating_point<T>::value>* = nullptr>
   void operator()(  column_device_view in,
                     mutable_column_device_view out,
                     hipStream_t stream)
   {      
      thrust::transform(rmm::exec_policy(stream)->on(stream),
                        thrust::make_counting_iterator(0),
                        thrust::make_counting_iterator(in.size()),
                        out.head<T>(), normalize_nans_and_zeros_lambda<T>{in});      
   }

   // if we get in here for anything but a float or double, that's a problem.
   template <typename T, std::enable_if_t<not std::is_floating_point<T>::value>* = nullptr>
   void operator()(  column_device_view in,
                     mutable_column_device_view out,
                     hipStream_t stream)
   {
      CUDF_FAIL("Unexpected non floating-point type.");      
   }   
};

} // end anonymous namespace

namespace cudf {
namespace detail {                                               

void normalize_nans_and_zeros(mutable_column_view in_out,
                              hipStream_t stream = 0)
{      
   if(in_out.size() == 0){
      return;
   }
   CUDF_EXPECTS(in_out.type() == data_type(FLOAT32) || in_out.type() == data_type(FLOAT64), "Expects float or double input");

   // wrapping the in_out data in a column_view so we can call the same lower level code.
   // that we use for the non in-place version.
   column_view input = in_out;

   // to device. unique_ptr which gets automatically cleaned up when we leave
   auto device_in = column_device_view::create(input);

   // from device. unique_ptr which gets automatically cleaned up when we leave.   
   auto device_out = mutable_column_device_view::create(in_out);

    // invoke the actual kernel.  
   cudf::experimental::type_dispatcher(input.type(), 
                                       normalize_nans_and_zeros_kernel_forwarder{},
                                       *device_in,
                                       *device_out,
                                       stream);
}

}  // namespace detail

/*
 * @brief Makes all NaNs and zeroes positive.
 *
 * Converts floating point values from @p input using the following rules:
 *        Convert  -NaN  -> NaN
 *        Convert  -0.0  -> 0.0
 *
 * @throws cudf::logic_error if column does not have floating point data type.
 * @param[in] column_view representing input data
 * @param[in] device_memory_resource allocator for allocating output data 
 *
 * @returns new column with the modified data
 */
std::unique_ptr<column> normalize_nans_and_zeros( column_view const& input,                                                                                                    
                                                  rmm::mr::device_memory_resource *mr)
{
   // output. copies the input
   std::unique_ptr<column> out = std::make_unique<column>(input, (hipStream_t)0, mr);   
   // from device. unique_ptr which gets automatically cleaned up when we leave.
   auto out_view = out->mutable_view();

   detail::normalize_nans_and_zeros(out_view, 0);

   return out;
}

/*
 * @brief Makes all Nans and zeroes positive.
 * 
 * Converts floating point values from @p in_out using the following rules:
 *        Convert  -NaN  -> NaN
 *        Convert  -0.0  -> 0.0
 *
 * @throws cudf::logic_error if column does not have floating point data type.
 * @param[in, out] mutable_column_view representing input data. data is processed in-place
 */
void normalize_nans_and_zeros(mutable_column_view& in_out)
{
   detail::normalize_nans_and_zeros(in_out, 0);
}

}  // namespace cudf

