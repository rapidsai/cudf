#include "hip/hip_runtime.h"
/*
 * Copyright 2018 BlazingDB, Inc.

 *     Copyright 2018 Cristhian Alberto Gonzales Castillo <cristhian@blazingdb.com>
 *     Copyright 2018 Alexander Ocsa <alexander@blazingdb.com>
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */
/*
 * Copyright (c) 2019-2020, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */
#include <cudf/column/column.hpp>
#include <cudf/column/column_device_view.cuh>
#include <cudf/column/column_factories.hpp>
#include <cudf/column/column_view.hpp>
#include <cudf/detail/concatenate.hpp>
#include <cudf/detail/copy.hpp>
#include <cudf/detail/null_mask.hpp>
#include <cudf/detail/nvtx/ranges.hpp>
#include <cudf/detail/replace.hpp>
#include <cudf/detail/utilities/cuda.cuh>
#include <cudf/dictionary/detail/update_keys.hpp>
#include <cudf/dictionary/dictionary_column_view.hpp>
#include <cudf/dictionary/dictionary_factories.hpp>
#include <cudf/replace.hpp>
#include <cudf/strings/detail/utilities.cuh>
#include <cudf/strings/detail/utilities.hpp>
#include <cudf/utilities/error.hpp>
#include <cudf/utilities/type_dispatcher.hpp>

#include <rmm/cuda_stream_view.hpp>
#include <rmm/device_scalar.hpp>

#include <thrust/find.h>

namespace {  // anonymous

static constexpr int BLOCK_SIZE = 256;

// return the new_value for output column at index `idx`
template <class T, bool replacement_has_nulls>
__device__ auto get_new_value(cudf::size_type idx,
                              const T* __restrict__ input_data,
                              const T* __restrict__ values_to_replace_begin,
                              const T* __restrict__ values_to_replace_end,
                              const T* __restrict__ d_replacement_values,
                              cudf::bitmask_type const* __restrict__ replacement_valid)
{
  auto found_ptr =
    thrust::find(thrust::seq, values_to_replace_begin, values_to_replace_end, input_data[idx]);
  T new_value{};
  bool output_is_valid{true};

  if (found_ptr != values_to_replace_end) {
    auto d    = thrust::distance(values_to_replace_begin, found_ptr);
    new_value = d_replacement_values[d];
    if (replacement_has_nulls) { output_is_valid = cudf::bit_is_set(replacement_valid, d); }
  } else {
    new_value = input_data[idx];
  }
  return thrust::make_pair(new_value, output_is_valid);
}

__device__ int get_new_string_value(cudf::size_type idx,
                                    cudf::column_device_view& input,
                                    cudf::column_device_view& values_to_replace,
                                    cudf::column_device_view& replacement_values)
{
  cudf::string_view input_string = input.element<cudf::string_view>(idx);
  int match                      = -1;
  for (int i = 0; i < values_to_replace.size(); i++) {
    cudf::string_view value_string = values_to_replace.element<cudf::string_view>(i);
    if (input_string == value_string) {
      match = i;
      break;
    }
  }
  return match;
}

/**
 * @brief Kernel which does the first pass of strings replace.
 *
 * It computes the output null_mask, null_count, and the offsets.
 *
 * @param input The input column to replace strings in.
 * @param values_to_replace The string values to replace.
 * @param replacement The replacement values.
 * @param offsets The column which will contain the offsets of the new string column
 * @param indices Temporary column used to store the replacement indices
 * @param output_valid The output null_mask
 * @param output_valid_count The output valid count
 */
template <bool input_has_nulls, bool replacement_has_nulls>
__global__ void replace_strings_first_pass(cudf::column_device_view input,
                                           cudf::column_device_view values_to_replace,
                                           cudf::column_device_view replacement,
                                           cudf::mutable_column_device_view offsets,
                                           cudf::mutable_column_device_view indices,
                                           cudf::bitmask_type* output_valid,
                                           cudf::size_type* __restrict__ output_valid_count)
{
  cudf::size_type nrows = input.size();
  cudf::size_type i     = blockIdx.x * blockDim.x + threadIdx.x;
  uint32_t active_mask  = 0xffffffff;
  active_mask           = __ballot_sync(active_mask, i < nrows);
  auto const lane_id{threadIdx.x % cudf::detail::warp_size};
  uint32_t valid_sum{0};

  while (i < nrows) {
    bool input_is_valid = true;

    if (input_has_nulls) input_is_valid = input.is_valid_nocheck(i);
    bool output_is_valid = input_is_valid;

    if (input_is_valid) {
      int result               = get_new_string_value(i, input, values_to_replace, replacement);
      cudf::string_view output = (result == -1) ? input.element<cudf::string_view>(i)
                                                : replacement.element<cudf::string_view>(result);
      offsets.data<cudf::size_type>()[i] = output.size_bytes();
      indices.data<cudf::size_type>()[i] = result;
      if (replacement_has_nulls && result != -1) {
        output_is_valid = replacement.is_valid_nocheck(result);
      }
    } else {
      offsets.data<cudf::size_type>()[i] = 0;
      indices.data<cudf::size_type>()[i] = -1;
    }

    uint32_t bitmask = __ballot_sync(active_mask, output_is_valid);
    if (0 == lane_id) {
      output_valid[cudf::word_index(i)] = bitmask;
      valid_sum += __popc(bitmask);
    }

    i += blockDim.x * gridDim.x;
    active_mask = __ballot_sync(active_mask, i < nrows);
  }

  // Compute total valid count for this block and add it to global count
  uint32_t block_valid_count = cudf::detail::single_lane_block_sum_reduce<BLOCK_SIZE, 0>(valid_sum);
  // one thread computes and adds to output_valid_count
  if (threadIdx.x == 0) { atomicAdd(output_valid_count, block_valid_count); }
}

/**
 * @brief Kernel which does the second pass of strings replace.
 *
 * It copies the string data needed from input and replacement into the new strings column chars
 * column.
 *
 * @param input The input column
 * @param replacement The replacement values
 * @param offsets The offsets column of the new strings column
 * @param strings The chars column of the new strings column
 * @param indices Temporary column used to store the replacement indices.
 */
template <bool input_has_nulls, bool replacement_has_nulls>
__global__ void replace_strings_second_pass(cudf::column_device_view input,
                                            cudf::column_device_view replacement,
                                            cudf::mutable_column_device_view offsets,
                                            cudf::mutable_column_device_view strings,
                                            cudf::mutable_column_device_view indices)
{
  cudf::size_type nrows = input.size();
  cudf::size_type i     = blockIdx.x * blockDim.x + threadIdx.x;

  while (i < nrows) {
    bool output_is_valid = true;
    bool input_is_valid  = true;
    cudf::size_type idx  = indices.element<cudf::size_type>(i);

    if (input_has_nulls) {
      input_is_valid  = input.is_valid_nocheck(i);
      output_is_valid = input_is_valid;
    }
    if (replacement_has_nulls && idx != -1) { output_is_valid = replacement.is_valid_nocheck(idx); }
    if (output_is_valid) {
      cudf::string_view output = (idx == -1) ? input.element<cudf::string_view>(i)
                                             : replacement.element<cudf::string_view>(idx);
      std::memcpy(strings.data<char>() + offsets.data<cudf::size_type>()[i],
                  output.data(),
                  output.size_bytes());
    }

    i += blockDim.x * gridDim.x;
  }
}

/**
 * @brief Kernel that replaces elements from `output_data` given the following
 *        rule: replace all `values_to_replace[i]` in [values_to_replace_begin`,
 *        `values_to_replace_end`) present in `output_data` with `d_replacement_values[i]`.
 *
 * @tparam input_has_nulls `true` if output column has valid mask, `false` otherwise
 * @tparam replacement_has_nulls `true` if replacement_values column has valid mask, `false`
 * otherwise The input_has_nulls and replacement_has_nulls template parameters allows us to
 * specialize this kernel for the different scenario for performance without writing different
 * kernel.
 *
 * @param[in] input_data Device array with the data to be modified
 * @param[in] input_valid Valid mask associated with input_data
 * @param[out] output_data Device array to store the data from input_data
 * @param[out] output_valid Valid mask associated with output_data
 * @param[out] output_valid_count #valid in output column
 * @param[in] nrows # rows in `output_data`
 * @param[in] values_to_replace_begin Device pointer to the beginning of the sequence
 * of old values to be replaced
 * @param[in] values_to_replace_end  Device pointer to the end of the sequence
 * of old values to be replaced
 * @param[in] d_replacement_values Device array with the new values
 * @param[in] replacement_valid Valid mask associated with d_replacement_values
 */
template <class T, bool input_has_nulls, bool replacement_has_nulls>
__global__ void replace_kernel(cudf::column_device_view input,
                               cudf::mutable_column_device_view output,
                               cudf::size_type* __restrict__ output_valid_count,
                               cudf::size_type nrows,
                               cudf::column_device_view values_to_replace,
                               cudf::column_device_view replacement)
{
  using Type = cudf::device_storage_type_t<T>;

  Type* __restrict__ output_data = output.data<Type>();

  cudf::size_type i = blockIdx.x * blockDim.x + threadIdx.x;

  uint32_t active_mask = 0xffffffff;
  active_mask          = __ballot_sync(active_mask, i < nrows);
  auto const lane_id{threadIdx.x % cudf::detail::warp_size};
  uint32_t valid_sum{0};

  while (i < nrows) {
    bool output_is_valid{true};
    bool input_is_valid{true};
    if (input_has_nulls) {
      input_is_valid  = input.is_valid_nocheck(i);
      output_is_valid = input_is_valid;
    }
    if (input_is_valid)
      thrust::tie(output_data[i], output_is_valid) = get_new_value<Type, replacement_has_nulls>(
        i,
        input.data<Type>(),
        values_to_replace.data<Type>(),
        values_to_replace.data<Type>() + values_to_replace.size(),
        replacement.data<Type>(),
        replacement.null_mask());

    /* output valid counts calculations*/
    if (input_has_nulls or replacement_has_nulls) {
      uint32_t bitmask = __ballot_sync(active_mask, output_is_valid);
      if (0 == lane_id) {
        output.set_mask_word(cudf::word_index(i), bitmask);
        valid_sum += __popc(bitmask);
      }
    }

    i += blockDim.x * gridDim.x;
    active_mask = __ballot_sync(active_mask, i < nrows);
  }
  if (input_has_nulls or replacement_has_nulls) {
    // Compute total valid count for this block and add it to global count
    uint32_t block_valid_count =
      cudf::detail::single_lane_block_sum_reduce<BLOCK_SIZE, 0>(valid_sum);
    // one thread computes and adds to output_valid_count
    if (threadIdx.x == 0) { atomicAdd(output_valid_count, block_valid_count); }
  }
}

/**
 * @brief Functor called by the `type_dispatcher` in order to invoke and instantiate
 *        `replace_kernel` with the appropriate data types.
 */
struct replace_kernel_forwarder {
  template <typename col_type, std::enable_if_t<cudf::is_fixed_width<col_type>()>* = nullptr>
  std::unique_ptr<cudf::column> operator()(cudf::column_view const& input_col,
                                           cudf::column_view const& values_to_replace,
                                           cudf::column_view const& replacement_values,
                                           rmm::cuda_stream_view stream,
                                           rmm::mr::device_memory_resource* mr)
  {
    rmm::device_scalar<cudf::size_type> valid_counter(0, stream);
    cudf::size_type* valid_count = valid_counter.data();

    auto replace = [&] {
      if (input_col.has_nulls())
        return replacement_values.has_nulls() ? replace_kernel<col_type, true, true>
                                              : replace_kernel<col_type, true, false>;
      else
        return replacement_values.has_nulls() ? replace_kernel<col_type, false, true>
                                              : replace_kernel<col_type, false, false>;
    }();

    auto output = [&] {
      auto const mask_allocation_policy = input_col.has_nulls() || replacement_values.has_nulls()
                                            ? cudf::mask_allocation_policy::ALWAYS
                                            : cudf::mask_allocation_policy::NEVER;
      return cudf::detail::allocate_like(
        input_col, input_col.size(), mask_allocation_policy, stream, mr);
    }();

    auto output_view = output->mutable_view();
    auto grid        = cudf::detail::grid_1d{output_view.size(), BLOCK_SIZE, 1};

    auto device_in                 = cudf::column_device_view::create(input_col);
    auto device_out                = cudf::mutable_column_device_view::create(output_view);
    auto device_values_to_replace  = cudf::column_device_view::create(values_to_replace);
    auto device_replacement_values = cudf::column_device_view::create(replacement_values);

    replace<<<grid.num_blocks, BLOCK_SIZE, 0, stream.value()>>>(*device_in,
                                                                *device_out,
                                                                valid_count,
                                                                output_view.size(),
                                                                *device_values_to_replace,
                                                                *device_replacement_values);

    if (output_view.nullable()) {
      output->set_null_count(output->size() - valid_counter.value(stream));
    }
    return output;
  }

  template <typename col_type, std::enable_if_t<not cudf::is_fixed_width<col_type>()>* = nullptr>
  std::unique_ptr<cudf::column> operator()(cudf::column_view const& input_col,
                                           cudf::column_view const& values_to_replace,
                                           cudf::column_view const& replacement_values,
                                           rmm::cuda_stream_view stream,
                                           rmm::mr::device_memory_resource* mr)
  {
    CUDF_FAIL("No specialization exists for this type");
  }
};

template <>
std::unique_ptr<cudf::column> replace_kernel_forwarder::operator()<cudf::string_view>(
  cudf::column_view const& input_col,
  cudf::column_view const& values_to_replace,
  cudf::column_view const& replacement_values,
  rmm::cuda_stream_view stream,
  rmm::mr::device_memory_resource* mr)
{
  rmm::device_scalar<cudf::size_type> valid_counter(0, stream);
  cudf::size_type* valid_count = valid_counter.data();

  auto replace_first  = replace_strings_first_pass<true, false>;
  auto replace_second = replace_strings_second_pass<true, false>;
  if (input_col.has_nulls()) {
    if (replacement_values.has_nulls()) {
      replace_first  = replace_strings_first_pass<true, true>;
      replace_second = replace_strings_second_pass<true, true>;
    }
  } else {
    if (replacement_values.has_nulls()) {
      replace_first  = replace_strings_first_pass<false, true>;
      replace_second = replace_strings_second_pass<false, true>;
    } else {
      replace_first  = replace_strings_first_pass<false, false>;
      replace_second = replace_strings_second_pass<false, false>;
    }
  }

  // Create new offsets column to use in kernel
  std::unique_ptr<cudf::column> sizes = cudf::make_numeric_column(
    cudf::data_type(cudf::type_id::INT32), input_col.size(), cudf::mask_state::UNALLOCATED, stream);
  std::unique_ptr<cudf::column> indices = cudf::make_numeric_column(
    cudf::data_type(cudf::type_id::INT32), input_col.size(), cudf::mask_state::UNALLOCATED, stream);

  auto sizes_view   = sizes->mutable_view();
  auto indices_view = indices->mutable_view();

  auto device_in                = cudf::column_device_view::create(input_col);
  auto device_values_to_replace = cudf::column_device_view::create(values_to_replace);
  auto device_replacement       = cudf::column_device_view::create(replacement_values);
  auto device_sizes             = cudf::mutable_column_device_view::create(sizes_view);
  auto device_indices           = cudf::mutable_column_device_view::create(indices_view);

  rmm::device_buffer valid_bits =
    cudf::detail::create_null_mask(input_col.size(), cudf::mask_state::UNINITIALIZED, stream, mr);

  // Call first pass kernel to get sizes in offsets
  cudf::detail::grid_1d grid{input_col.size(), BLOCK_SIZE, 1};
  replace_first<<<grid.num_blocks, BLOCK_SIZE, 0, stream.value()>>>(
    *device_in,
    *device_values_to_replace,
    *device_replacement,
    *device_sizes,
    *device_indices,
    reinterpret_cast<cudf::bitmask_type*>(valid_bits.data()),
    valid_count);

  std::unique_ptr<cudf::column> offsets = cudf::strings::detail::make_offsets_child_column(
    sizes_view.begin<int32_t>(), sizes_view.end<int32_t>(), stream, mr);
  auto offsets_view   = offsets->mutable_view();
  auto device_offsets = cudf::mutable_column_device_view::create(offsets_view);
  int32_t size;
  CUDA_TRY(hipMemcpyAsync(
    &size, offsets_view.end<int32_t>() - 1, sizeof(int32_t), hipMemcpyDefault, stream.value()));
  stream.synchronize();

  // Allocate chars array and output null mask
  cudf::size_type null_count                 = input_col.size() - valid_counter.value(stream);
  std::unique_ptr<cudf::column> output_chars = cudf::strings::detail::create_chars_child_column(
    input_col.size(), null_count, size, stream, mr);

  auto output_chars_view = output_chars->mutable_view();
  auto device_chars      = cudf::mutable_column_device_view::create(output_chars_view);

  replace_second<<<grid.num_blocks, BLOCK_SIZE, 0, stream.value()>>>(
    *device_in, *device_replacement, *device_offsets, *device_chars, *device_indices);

  return cudf::make_strings_column(input_col.size(),
                                   std::move(offsets),
                                   std::move(output_chars),
                                   null_count,
                                   std::move(valid_bits),
                                   stream,
                                   mr);
}

template <>
std::unique_ptr<cudf::column> replace_kernel_forwarder::operator()<cudf::dictionary32>(
  cudf::column_view const& input_col,
  cudf::column_view const& values_to_replace,
  cudf::column_view const& replacement_values,
  rmm::cuda_stream_view stream,
  rmm::mr::device_memory_resource* mr)
{
  auto input        = cudf::dictionary_column_view(input_col);
  auto values       = cudf::dictionary_column_view(values_to_replace);
  auto replacements = cudf::dictionary_column_view(replacement_values);

  auto matched_input = [&] {
    auto new_keys = cudf::detail::concatenate({values.keys(), replacements.keys()}, stream);
    return cudf::dictionary::detail::add_keys(input, new_keys->view(), stream, mr);
  }();
  auto matched_view   = cudf::dictionary_column_view(matched_input->view());
  auto matched_values = cudf::dictionary::detail::set_keys(values, matched_view.keys(), stream);
  auto matched_replacements =
    cudf::dictionary::detail::set_keys(replacements, matched_view.keys(), stream);

  auto indices_type = matched_view.indices().type();
  auto new_indices  = cudf::type_dispatcher(
    indices_type,
    replace_kernel_forwarder{},
    matched_view.get_indices_annotated(),
    cudf::dictionary_column_view(matched_values->view()).indices(),
    cudf::dictionary_column_view(matched_replacements->view()).get_indices_annotated(),
    stream,
    mr);
  auto null_count     = new_indices->null_count();
  auto contents       = new_indices->release();
  auto indices_column = std::make_unique<cudf::column>(
    indices_type, input.size(), *(contents.data.release()), rmm::device_buffer{0, stream, mr}, 0);
  std::unique_ptr<cudf::column> keys_column(std::move(matched_input->release().children.back()));
  return cudf::make_dictionary_column(std::move(keys_column),
                                      std::move(indices_column),
                                      std::move(*(contents.null_mask.release())),
                                      null_count);
}

}  // end anonymous namespace

namespace cudf {
namespace detail {
std::unique_ptr<cudf::column> find_and_replace_all(cudf::column_view const& input_col,
                                                   cudf::column_view const& values_to_replace,
                                                   cudf::column_view const& replacement_values,
                                                   rmm::cuda_stream_view stream,
                                                   rmm::mr::device_memory_resource* mr)
{
  CUDF_EXPECTS(values_to_replace.size() == replacement_values.size(),
               "values_to_replace and replacement_values size mismatch.");

  CUDF_EXPECTS(
    input_col.type() == values_to_replace.type() && input_col.type() == replacement_values.type(),
    "Columns type mismatch");
  CUDF_EXPECTS(values_to_replace.has_nulls() == false, "values_to_replace must not have nulls");

  if (0 == input_col.size() || 0 == values_to_replace.size() || 0 == replacement_values.size()) {
    return std::make_unique<cudf::column>(input_col);
  }

  return cudf::type_dispatcher(input_col.type(),
                               replace_kernel_forwarder{},
                               input_col,
                               values_to_replace,
                               replacement_values,
                               stream,
                               mr);
}

}  // namespace detail

/**
 * @brief Replace elements from `input_col` according to the mapping `values_to_replace` to
 *        `replacement_values`, that is, replace all `values_to_replace[i]` present in `input_col`
 *        with `replacement_values[i]`.
 *
 * @param[in] col column_view of the data to be modified
 * @param[in] values_to_replace column_view of the old values to be replaced
 * @param[in] replacement_values column_view of the new values
 *
 * @returns output cudf::column with the modified data
 */
std::unique_ptr<cudf::column> find_and_replace_all(cudf::column_view const& input_col,
                                                   cudf::column_view const& values_to_replace,
                                                   cudf::column_view const& replacement_values,
                                                   rmm::mr::device_memory_resource* mr)
{
  return cudf::detail::find_and_replace_all(
    input_col, values_to_replace, replacement_values, rmm::cuda_stream_default, mr);
}
}  // namespace cudf
