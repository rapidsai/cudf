/*
 * Copyright (c) 2021-2022, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <cudf/detail/copy.hpp>
#include <cudf/detail/copy_if_else.cuh>
#include <cudf/detail/iterator.cuh>
#include <cudf/scalar/scalar.hpp>
#include <cudf/strings/detail/copy_if_else.cuh>
#include <cudf/strings/string_view.cuh>
#include <cudf/types.hpp>
#include <cudf/utilities/traits.hpp>

#include <rmm/cuda_stream_view.hpp>

#include <thrust/binary_search.h>
#include <thrust/execution_policy.h>
#include <thrust/iterator/transform_iterator.h>

namespace cudf {
namespace detail {

namespace {

/**
 * @brief Common filter function to convert index values into copy-if-else left/right result.
 *
 * The offset position is used to identify which segment to copy from.
 */
struct segmented_shift_filter {
  device_span<size_type const> const segment_offsets;
  size_type const offset;

  __device__ bool operator()(size_type const i) const
  {
    auto const segment_bound_idx =
      thrust::upper_bound(thrust::seq, segment_offsets.begin(), segment_offsets.end(), i) -
      (offset > 0);
    auto const left_idx  = *segment_bound_idx + (offset < 0 ? offset : 0);
    auto const right_idx = *segment_bound_idx + (offset > 0 ? offset : 0);
    return not(left_idx <= i and i < right_idx);
  };
};

template <typename T, typename Enable = void>
struct segmented_shift_functor {
  template <typename... Args>
  std::unique_ptr<column> operator()(Args&&...)
  {
    CUDF_FAIL("Unsupported type for segmented_shift.");
  }
};

/**
 * @brief Segmented shift specialization for representation layout compatible types.
 */
template <typename T>
struct segmented_shift_functor<T, std::enable_if_t<is_rep_layout_compatible<T>()>> {
  std::unique_ptr<column> operator()(column_view const& segmented_values,
                                     device_span<size_type const> segment_offsets,
                                     size_type offset,
                                     scalar const& fill_value,
                                     rmm::cuda_stream_view stream,
                                     rmm::mr::device_memory_resource* mr)
  {
    auto values_device_view = column_device_view::create(segmented_values, stream);
    bool nullable           = not fill_value.is_valid(stream) or segmented_values.nullable();
    auto input_iterator     = cudf::detail::make_optional_iterator<T>(
                            *values_device_view, nullate::DYNAMIC{segmented_values.has_nulls()}) -
                          offset;
    auto fill_iterator = cudf::detail::make_optional_iterator<T>(fill_value, nullate::YES{});
    return copy_if_else(nullable,
                        input_iterator,
                        input_iterator + segmented_values.size(),
                        fill_iterator,
                        segmented_shift_filter{segment_offsets, offset},
                        segmented_values.type(),
                        stream,
                        mr);
  }
};

/**
 * @brief Segmented shift specialization for `string_view`.
 */
template <>
struct segmented_shift_functor<string_view> {
  std::unique_ptr<column> operator()(column_view const& segmented_values,
                                     device_span<size_type const> segment_offsets,
                                     size_type offset,
                                     scalar const& fill_value,
                                     rmm::cuda_stream_view stream,
                                     rmm::mr::device_memory_resource* mr)
  {
    auto values_device_view = column_device_view::create(segmented_values, stream);
    auto input_iterator     = make_optional_iterator<cudf::string_view>(
                            *values_device_view, nullate::DYNAMIC{segmented_values.has_nulls()}) -
                          offset;
    auto fill_iterator = make_optional_iterator<cudf::string_view>(fill_value, nullate::YES{});
    return strings::detail::copy_if_else(input_iterator,
                                         input_iterator + segmented_values.size(),
                                         fill_iterator,
                                         segmented_shift_filter{segment_offsets, offset},
                                         stream,
                                         mr);
  }
};

/**
 * @brief Functor to instantiate the specializations for segmented shift and
 * forward arguments.
 */
struct segmented_shift_functor_forwarder {
  template <typename T>
  std::unique_ptr<column> operator()(column_view const& segmented_values,
                                     device_span<size_type const> segment_offsets,
                                     size_type offset,
                                     scalar const& fill_value,
                                     rmm::cuda_stream_view stream,
                                     rmm::mr::device_memory_resource* mr)
  {
    segmented_shift_functor<T> shifter;
    return shifter(segmented_values, segment_offsets, offset, fill_value, stream, mr);
  }
};

}  // namespace

std::unique_ptr<column> segmented_shift(column_view const& segmented_values,
                                        device_span<size_type const> segment_offsets,
                                        size_type offset,
                                        scalar const& fill_value,
                                        rmm::cuda_stream_view stream,
                                        rmm::mr::device_memory_resource* mr)
{
  if (segmented_values.is_empty()) { return empty_like(segmented_values); }
  if (offset == 0) { return std::make_unique<column>(segmented_values, stream, mr); };

  return type_dispatcher<dispatch_storage_type>(segmented_values.type(),
                                                segmented_shift_functor_forwarder{},
                                                segmented_values,
                                                segment_offsets,
                                                offset,
                                                fill_value,
                                                stream,
                                                mr);
}

}  // namespace detail
}  // namespace cudf
