#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */
 
#include <cudf/cudf.h>
#include <cudf/copying.hpp>
#include <cudf/filling.hpp>

#include <copying/copy_range.cuh>
#include <filling/scalar_factory.cuh>

#include <vector>

namespace cudf {

namespace detail {

  void shift(
    gdf_column *out_column,
    gdf_column const &in_column,
    gdf_index_type periods,
    gdf_scalar const fill_value
  )
  {
    gdf_index_type in_start = 0;
    gdf_index_type out_start = periods;
    gdf_index_type out_end = out_column->size;
    gdf_index_type fill_start = 0;
    gdf_index_type fill_end = out_start;
  
    if (periods < 0) {
      in_start = -periods;
      out_start = 0;
      out_end = out_column->size + periods;
      fill_start = out_end;
      fill_end = out_column->size;
    }

    if (std::abs(periods) < out_column->size) {
      detail::copy_range(out_column,
                         detail::column_range_factory{in_column, in_start},
                         out_start,
                         out_end);
    } else {
      fill_start = 0;
      fill_end = out_column->size;
    }

    detail::copy_range(out_column,
                       detail::scalar_factory{fill_value},
                       fill_start,
                       fill_end);
  }

}; // namespace detail

gdf_column shift(
  const gdf_column& in,
  gdf_index_type periods,
  const gdf_scalar fill_value
)
{
  if (in.size == 0 || periods == 0) {
    return copy(in);
  }

  gdf_column out;

  if (fill_value.is_valid == false || is_nullable(in)) {
    out = allocate_like(in, ALWAYS);
    auto valid_size = gdf_valid_allocation_size(out.size);
    out.null_count = 0;
    CUDA_TRY(hipMemset(out.valid, 0xff, valid_size));
  } else {
    out = allocate_like(in, NEVER);
  }

  detail::shift(&out, in, periods, fill_value);

  return out;
}

}; // namespace cudf
