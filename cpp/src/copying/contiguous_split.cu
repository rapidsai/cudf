#include "hip/hip_runtime.h"
#include <cudf/cudf.h>
#include <cudf/column/column_view.hpp>
#include <cudf/table/table_view.hpp>
#include <cudf/column/column_device_view.cuh>
#include <cudf/copying.hpp>
#include <cudf/detail/utilities/cuda.cuh>
#include <cudf/utilities/bit.hpp>

#include <numeric>

namespace cudf {

namespace experimental {

namespace detail {

namespace {

/**
 * @brief Copies contents of `in` to `out`.  Copies validity if present
 * but does not compute null count.
 *  
 * @param in column_view to copy from
 * @param out mutable_column_view to copy to.
 */
template <size_type block_size, typename T, bool has_validity>
__launch_bounds__(block_size)
__global__
void copy_in_place_kernel( column_device_view const in,
                           mutable_column_device_view out)
{
   const size_type tid = threadIdx.x + blockIdx.x * block_size;
   const int warp_id = tid / cudf::experimental::detail::warp_size;
   const size_type warps_per_grid = gridDim.x * block_size / cudf::experimental::detail::warp_size;      

   // begin/end indices for the column data
   size_type begin = 0;
   size_type end = in.size();
   // warp indices.  since 1 warp == 32 threads == sizeof(bit_mask_t) * 8,
   // each warp will process one (32 bit) of the validity mask via
   // __ballot_sync()
   size_type warp_begin = cudf::word_index(begin);
   size_type warp_end = cudf::word_index(end-1);      

   // lane id within the current warp   
   const int lane_id = threadIdx.x % cudf::experimental::detail::warp_size;
   
   // current warp.
   size_type warp_cur = warp_begin + warp_id;   
   size_type index = tid;
   while(warp_cur <= warp_end){
      bool in_range = (index >= begin && index < end);
            
      bool valid = true;
      if(has_validity){
         valid = in_range && in.is_valid(index);
      }
      if(in_range){
         out.element<T>(index) = in.element<T>(index);
      }
      
      // update validity      
      if(has_validity){
         // the final validity mask for this warp 
         int warp_mask = __ballot_sync(0xFFFF'FFFF, valid && in_range);
         // only one guy in the warp needs to update the mask and count
         if(lane_id == 0){            
            out.set_mask_word(warp_cur, warp_mask);            
         }
      }            

      // next grid
      warp_cur += warps_per_grid;
      index += block_size * gridDim.x;
   }      
}

/**
 * @brief Functor called by the `type_dispatcher` to incrementally compute total
 * memory buffer size needed to allocate a contiguous copy of all columns within
 * a source table. 
 */
struct column_buffer_size_functor {
   // align all column size allocations to this boundary so that all output column buffers
   // start at that alignment.
   static constexpr size_t split_align = 64;

   template <typename T, std::enable_if_t<not is_fixed_width<T>()>* = nullptr>
   std::pair<size_t, size_t> operator()(std::pair<size_t, size_t> sizes, column_view const& c)
   {
      CUDF_FAIL("contiguous_split for strings not implemented yet");
      return std::pair<size_t, size_t>(0, 0);
   }

   template <typename T, std::enable_if_t<is_fixed_width<T>()>* = nullptr>
   std::pair<size_t, size_t> operator()(std::pair<size_t, size_t> sizes, column_view const& c)
   {      
      size_t data_size = sizes.first + cudf::util::round_up_safe(c.size() * sizeof(T), split_align);  
      size_t validity_size = sizes.second + (c.nullable() ? cudf::bitmask_allocation_size_bytes(c.size(), split_align) : 0);
      return std::pair<size_t, size_t>(data_size, validity_size);
   }
};

/**
 * @brief Functor called by the `type_dispatcher` to copy a column into a contiguous
 * buffer of output memory. 
 * 
 * Used for copying each column in a source table into one contiguous buffer of memory.
 */
struct column_copy_functor {
   template <typename T, std::enable_if_t<not is_fixed_width<T>()>* = nullptr>
   void operator()(column_view const& in, char*& dst, std::vector<column_view>& out_cols)
   {
      CUDF_FAIL("contiguous_split for strings not implemented yet");
   }

   template <typename T, std::enable_if_t<is_fixed_width<T>()>* = nullptr>
   void operator()(column_view const& in, char*& dst, std::vector<column_view>& out_cols)
   {
      // there's some unnecessary recomputation of sizes happening here, but it really shouldn't affect much.      
      auto sizes = column_buffer_size_functor{}.operator()<T>(std::pair<size_t, size_t>(0, 0), in);
      size_t data_size = sizes.first;
      size_t validity_size = sizes.second;
   
      // outgoing pointers
      char* data = dst;
      bitmask_type* validity = validity_size == 0 ? nullptr : reinterpret_cast<bitmask_type*>(dst + data_size);

      // increment working buffer
      dst += (data_size + validity_size);

      // custom copy kernel (which should probably just be an in-place copy() function in cudf.
      cudf::size_type num_els = cudf::util::round_up_safe(in.size(), cudf::experimental::detail::warp_size);
      constexpr int block_size = 256;
      cudf::experimental::detail::grid_1d grid{num_els, block_size, 1};
      
      // so there's a significant performance issue that comes up. our incoming column_view objects
      // are the result of a slice.  because of this, they have an UNKNOWN_NULL_COUNT.  because of that,
      // calling column_device_view::create() will cause a recompute of the count, which ends up being
      // extremely slow because a.) the typical use case here will involve huge numbers of calls and
      // b.) the count recompute involves tons of device allocs and memcopies.
      //
      // so to get around this, I am manually constructing a fake-ish view here where the null
      // count is arbitrarily bashed to 0.            
      //            
      // Remove this hack once rapidsai/cudf#3600 is fixed.
      column_view   in_wrapped{in.type(), in.size(), in.head<T>(), 
                               in.null_mask(), in.null_mask() == nullptr ? UNKNOWN_NULL_COUNT : 0,
                               in.offset() };
      mutable_column_view  mcv{in.type(), in.size(), data, 
                               validity, validity == nullptr ? UNKNOWN_NULL_COUNT : 0 };      
      if(in.nullable()){               
         copy_in_place_kernel<block_size, T, true><<<grid.num_blocks, block_size, 0, 0>>>(
                           *column_device_view::create(in_wrapped), 
                           *mutable_column_device_view::create(mcv));         
      } else {
         copy_in_place_kernel<block_size, T, false><<<grid.num_blocks, block_size, 0, 0>>>(
                           *column_device_view::create(in_wrapped), 
                           *mutable_column_device_view::create(mcv));
      }
      mcv.set_null_count(cudf::UNKNOWN_NULL_COUNT);                 

      out_cols.push_back(mcv);
   }
};

/**
 * @brief Creates a contiguous_split_result object which contains a deep-copy of the input
 * table_view into a single contiguous block of memory. 
 * 
 * The table_view contained within the contiguous_split_result will pass an expect_tables_equal()
 * call with the input table.  The memory referenced by the table_view and its internal column_views
 * is entirely contained in single block of memory.
 */
contiguous_split_result alloc_and_copy(cudf::table_view const& t, rmm::mr::device_memory_resource* mr, hipStream_t stream)      
{   
   // compute sizes  
   auto sizes = std::accumulate(t.begin(), t.end(), std::pair<size_t, size_t>(0, 0), 
      [](std::pair<size_t, size_t> sizes, cudf::column_view const& c){
         return cudf::experimental::type_dispatcher(c.type(), column_buffer_size_functor{}, sizes, c);
      });
   size_t data_size = sizes.first;
   size_t validity_size = sizes.second;   

   // allocate 
   auto device_buf = std::make_unique<rmm::device_buffer>(data_size + validity_size, stream, mr);
   char *buf = static_cast<char*>(device_buf->data());

   // copy
   std::vector<column_view> out_cols;
   out_cols.reserve(t.num_columns());
   std::for_each(t.begin(), t.end(), [&out_cols, &buf](cudf::column_view const& c){
      cudf::experimental::type_dispatcher(c.type(), column_copy_functor{}, c, buf, out_cols);
   });
   
   return contiguous_split_result{cudf::table_view{out_cols}, std::move(device_buf)};
}

}; // anonymous namespace

std::vector<contiguous_split_result> contiguous_split(cudf::table_view const& input,
                                                      std::vector<size_type> const& splits,
                                                      rmm::mr::device_memory_resource* mr,
                                                      hipStream_t stream)
{          
   auto subtables = cudf::experimental::split(input, splits);      

   std::vector<contiguous_split_result> result;
   std::transform(subtables.begin(), subtables.end(), std::back_inserter(result), [mr, stream](table_view const& t) { 
      return alloc_and_copy(t, mr, stream);
   });

   return result;
}

}; // namespace detail

std::vector<contiguous_split_result> contiguous_split(cudf::table_view const& input,
                                                      std::vector<size_type> const& splits,
                                                      rmm::mr::device_memory_resource* mr)
{    
   return cudf::experimental::detail::contiguous_split(input, splits, mr, (hipStream_t)0);   
}

}; // namespace experimental

}; // namespace cudf
