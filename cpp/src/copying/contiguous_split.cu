#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019-2024, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <cudf/column/column_device_view.cuh>
#include <cudf/column/column_view.hpp>
#include <cudf/contiguous_split.hpp>
#include <cudf/detail/contiguous_split.hpp>
#include <cudf/detail/copy.hpp>
#include <cudf/detail/iterator.cuh>
#include <cudf/detail/null_mask.hpp>
#include <cudf/detail/utilities/cuda.cuh>
#include <cudf/detail/utilities/vector_factories.hpp>
#include <cudf/lists/lists_column_view.hpp>
#include <cudf/structs/structs_column_view.hpp>
#include <cudf/table/table_view.hpp>
#include <cudf/utilities/bit.hpp>
#include <cudf/utilities/default_stream.hpp>

#include <rmm/cuda_stream_view.hpp>
#include <rmm/exec_policy.hpp>

#include <cuda/functional>
#include <thrust/binary_search.h>
#include <thrust/execution_policy.h>
#include <thrust/for_each.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/iterator/discard_iterator.h>
#include <thrust/iterator/iterator_categories.h>
#include <thrust/iterator/transform_iterator.h>
#include <thrust/pair.h>
#include <thrust/reduce.h>
#include <thrust/scan.h>
#include <thrust/transform.h>
#include <thrust/tuple.h>

#include <cstddef>
#include <numeric>
#include <stdexcept>

namespace cudf {
namespace {

// Align all column size allocations to this boundary so that all output column buffers
// start at that alignment.
static constexpr std::size_t split_align = 64;

// The size that contiguous split uses internally as the GPU unit of work.
// The number of `desired_batch_size` batches equals the number of CUDA blocks
// that will be used for the main kernel launch (`copy_partitions`).
static constexpr std::size_t desired_batch_size = 1 * 1024 * 1024;

/**
 * @brief Struct which contains information on a source buffer.
 *
 * The definition of "buffer" used throughout this module is a component piece of a
 * cudf column. So for example, a fixed-width column with validity would have 2 associated
 * buffers : the data itself and the validity buffer.  contiguous_split operates by breaking
 * each column up into it's individual components and copying each one as a separate kernel
 * block.
 */
struct src_buf_info {
  src_buf_info(cudf::type_id _type,
               int const* _offsets,
               int _offset_stack_pos,
               int _parent_offsets_index,
               bool _is_validity,
               size_type _column_offset)
    : type(_type),
      offsets(_offsets),
      offset_stack_pos(_offset_stack_pos),
      parent_offsets_index(_parent_offsets_index),
      is_validity(_is_validity),
      column_offset(_column_offset)
  {
  }

  cudf::type_id type;
  int const* offsets;        // a pointer to device memory offsets if I am an offset buffer
  int offset_stack_pos;      // position in the offset stack buffer
  int parent_offsets_index;  // immediate parent that has offsets, or -1 if none
  bool is_validity;          // if I am a validity buffer
  size_type column_offset;   // offset in the case of a sliced column
};

/**
 * @brief Struct which contains information on a destination buffer.
 *
 * Similar to src_buf_info, dst_buf_info contains information on a destination buffer we
 * are going to copy to.  If we have N input buffers (which come from X columns), and
 * M partitions, then we have N*M destination buffers.
 */
struct dst_buf_info {
  // constant across all copy commands for this buffer
  std::size_t buf_size;  // total size of buffer, including padding
  int num_elements;      // # of elements to be copied
  int element_size;      // size of each element in bytes
  int num_rows;  // # of rows to be copied(which may be different from num_elements in the case of
                 // validity or offset buffers)

  int src_element_index;   // element index to start reading from my associated source buffer
  std::size_t dst_offset;  // my offset into the per-partition allocation
  int value_shift;         // amount to shift values down by (for offset buffers)
  int bit_shift;           // # of bits to shift right by (for validity buffers)
  size_type valid_count;   // validity count for this block of work

  int src_buf_index;  // source buffer index
  int dst_buf_index;  // destination buffer index
};

/**
 * @brief Copy a single buffer of column data, shifting values (for offset columns),
 * and validity (for validity buffers) as necessary.
 *
 * Copies a single partition of a source column buffer to a destination buffer. Shifts
 * element values by value_shift in the case of a buffer of offsets (value_shift will
 * only ever be > 0 in that case).  Shifts elements bitwise by bit_shift in the case of
 * a validity buffer (bit_shift will only ever be > 0 in that case).  This function assumes
 * value_shift and bit_shift will never be > 0 at the same time.
 *
 * This function expects:
 * - src may be a misaligned address
 * - dst must be an aligned address
 *
 * This function always does the ALU work related to value_shift and bit_shift because it is
 * entirely memory-bandwidth bound.
 *
 * @param dst Destination buffer
 * @param src Source buffer
 * @param t Thread index
 * @param num_elements Number of elements to copy
 * @param element_size Size of each element in bytes
 * @param src_element_index Element index to start copying at
 * @param stride Size of the kernel block
 * @param value_shift Shift incoming 4-byte offset values down by this amount
 * @param bit_shift Shift incoming data right by this many bits
 * @param num_rows Number of rows being copied
 * @param valid_count Optional pointer to a value to store count of set bits
 */
template <int block_size>
__device__ void copy_buffer(uint8_t* __restrict__ dst,
                            uint8_t const* __restrict__ src,
                            int t,
                            std::size_t num_elements,
                            std::size_t element_size,
                            std::size_t src_element_index,
                            uint32_t stride,
                            int value_shift,
                            int bit_shift,
                            std::size_t num_rows,
                            size_type* valid_count)
{
  src += (src_element_index * element_size);

  size_type thread_valid_count = 0;

  // handle misalignment. read 16 bytes in 4 byte reads. write in a single 16 byte store.
  std::size_t const num_bytes = num_elements * element_size;
  // how many bytes we're misaligned from 4-byte alignment
  uint32_t const ofs = reinterpret_cast<uintptr_t>(src) % 4;
  std::size_t pos    = t * 16;
  stride *= 16;
  while (pos + 20 <= num_bytes) {
    // read from the nearest aligned address.
    uint32_t const* in32 = reinterpret_cast<uint32_t const*>((src + pos) - ofs);
    uint4 v              = uint4{in32[0], in32[1], in32[2], in32[3]};
    if (ofs || bit_shift) {
      v.x = __funnelshift_r(v.x, v.y, ofs * 8 + bit_shift);
      v.y = __funnelshift_r(v.y, v.z, ofs * 8 + bit_shift);
      v.z = __funnelshift_r(v.z, v.w, ofs * 8 + bit_shift);
      v.w = __funnelshift_r(v.w, in32[4], ofs * 8 + bit_shift);
    }
    v.x -= value_shift;
    v.y -= value_shift;
    v.z -= value_shift;
    v.w -= value_shift;
    reinterpret_cast<uint4*>(dst)[pos / 16] = v;
    if (valid_count) {
      thread_valid_count += (__popc(v.x) + __popc(v.y) + __popc(v.z) + __popc(v.w));
    }
    pos += stride;
  }

  // copy trailing bytes
  if (t == 0) {
    std::size_t remainder;
    if (num_bytes < 16) {
      remainder = num_bytes;
    } else {
      std::size_t const last_bracket = (num_bytes / 16) * 16;
      remainder                      = num_bytes - last_bracket;
      if (remainder < 4) {
        // we had less than 20 bytes for the last possible 16 byte copy, so copy 16 + the extra
        remainder += 16;
      }
    }

    // if we're performing a value shift (offsets), or a bit shift (validity) the # of bytes and
    // alignment must be a multiple of 4. value shifting and bit shifting are mutually exclusive
    // and will never both be true at the same time.
    if (value_shift || bit_shift) {
      std::size_t idx = (num_bytes - remainder) / 4;
      uint32_t v = remainder > 0 ? (reinterpret_cast<uint32_t const*>(src)[idx] - value_shift) : 0;

      constexpr size_type rows_per_element = 32;
      auto const have_trailing_bits = ((num_elements * rows_per_element) - num_rows) < bit_shift;
      while (remainder) {
        // if we're at the very last word of a validity copy, we do not always need to read the next
        // word to get the final trailing bits.
        auto const read_trailing_bits = bit_shift > 0 && remainder == 4 && have_trailing_bits;
        uint32_t const next           = (read_trailing_bits || remainder > 4)
                                          ? (reinterpret_cast<uint32_t const*>(src)[idx + 1] - value_shift)
                                          : 0;

        uint32_t const val = (v >> bit_shift) | (next << (32 - bit_shift));
        if (valid_count) { thread_valid_count += __popc(val); }
        reinterpret_cast<uint32_t*>(dst)[idx] = val;
        v                                     = next;
        idx++;
        remainder -= 4;
      }
    } else {
      while (remainder) {
        std::size_t const idx = num_bytes - remainder--;
        uint32_t const val    = reinterpret_cast<uint8_t const*>(src)[idx];
        if (valid_count) { thread_valid_count += __popc(val); }
        reinterpret_cast<uint8_t*>(dst)[idx] = val;
      }
    }
  }

  if (valid_count) {
    if (num_bytes == 0) {
      if (!t) { *valid_count = 0; }
    } else {
      using BlockReduce = hipcub::BlockReduce<size_type, block_size>;
      __shared__ typename BlockReduce::TempStorage temp_storage;
      size_type block_valid_count{BlockReduce(temp_storage).Sum(thread_valid_count)};
      if (!t) {
        // we may have copied more bits than there are actual rows in the output.
        // so we need to subtract off the count of any bits that shouldn't have been
        // considered during the copy step.
        std::size_t const max_row    = (num_bytes * 8);
        std::size_t const slack_bits = max_row > num_rows ? max_row - num_rows : 0;
        auto const slack_mask        = set_most_significant_bits(slack_bits);
        if (slack_mask > 0) {
          uint32_t const last_word = reinterpret_cast<uint32_t*>(dst + (num_bytes - 4))[0];
          block_valid_count -= __popc(last_word & slack_mask);
        }
        *valid_count = block_valid_count;
      }
    }
  }
}

/**
 * @brief Kernel which copies data from multiple source buffers to multiple
 * destination buffers.
 *
 * When doing a contiguous_split on X columns comprising N total internal buffers
 * with M splits, we end up having to copy N*M source/destination buffer pairs.
 * These logical copies are further subdivided to distribute the amount of work
 * to be done as evenly as possible across the multiprocessors on the device.
 * This kernel is arranged such that each block copies 1 source/destination pair.
 *
 * @param index_to_buffer A function that given a `buf_index` returns the destination buffer
 * @param src_bufs Input source buffers
 * @param buf_info Information on the range of values to be copied for each destination buffer
 */
template <int block_size, typename IndexToDstBuf>
CUDF_KERNEL void copy_partitions(IndexToDstBuf index_to_buffer,
                                 uint8_t const** src_bufs,
                                 dst_buf_info* buf_info)
{
  auto const buf_index     = blockIdx.x;
  auto const src_buf_index = buf_info[buf_index].src_buf_index;

  // copy, shifting offsets and validity bits as needed
  copy_buffer<block_size>(
    index_to_buffer(buf_index) + buf_info[buf_index].dst_offset,
    src_bufs[src_buf_index],
    threadIdx.x,
    buf_info[buf_index].num_elements,
    buf_info[buf_index].element_size,
    buf_info[buf_index].src_element_index,
    blockDim.x,
    buf_info[buf_index].value_shift,
    buf_info[buf_index].bit_shift,
    buf_info[buf_index].num_rows,
    buf_info[buf_index].valid_count > 0 ? &buf_info[buf_index].valid_count : nullptr);
}

// The block of functions below are all related:
//
// compute_offset_stack_size()
// setup_src_buf_data()
// count_src_bufs()
// setup_source_buf_info()
// build_output_columns()
//
// Critically, they all traverse the hierarchy of source columns and their children
// in a specific order to guarantee they produce various outputs in a consistent
// way.  For example, setup_src_buf_info() produces a series of information
// structs that must appear in the same order that setup_src_buf_data() produces
// buffers.
//
// So please be careful if you change the way in which these functions and
// functors traverse the hierarchy.

/**
 * @brief Returns whether or not the specified type is a column that contains offsets.
 */
bool is_offset_type(type_id id) { return (id == type_id::STRING or id == type_id::LIST); }

/**
 * @brief Compute total device memory stack size needed to process nested
 * offsets per-output buffer.
 *
 * When determining the range of rows to be copied for each output buffer
 * we have to recursively apply the stack of offsets from our parent columns
 * (lists or strings).  We want to do this computation on the gpu because offsets
 * are stored in device memory.  However we don't want to do recursion on the gpu, so
 * each destination buffer gets a "stack" of space to work with equal in size to
 * it's offset nesting depth.  This function computes the total size of all of those
 * stacks.
 *
 * This function is called recursively in the case of nested types.
 *
 * @param begin Beginning of input columns
 * @param end End of input columns
 * @param offset_depth Current offset nesting depth
 *
 * @returns Total offset stack size needed for this range of columns
 */
template <typename InputIter>
std::size_t compute_offset_stack_size(InputIter begin, InputIter end, int offset_depth = 0)
{
  return std::accumulate(begin, end, 0, [offset_depth](auto stack_size, column_view const& col) {
    auto const num_buffers = 1 + (col.nullable() ? 1 : 0);
    return stack_size + (offset_depth * num_buffers) +
           compute_offset_stack_size(
             col.child_begin(), col.child_end(), offset_depth + is_offset_type(col.type().id()));
  });
}

/**
 * @brief Retrieve all buffers for a range of source columns.
 *
 * Retrieve the individual buffers that make up a range of input columns.
 *
 * This function is called recursively in the case of nested types.
 *
 * @param begin Beginning of input columns
 * @param end End of input columns
 * @param out_buf Iterator into output buffer infos
 *
 * @returns next output buffer iterator
 */
template <typename InputIter, typename OutputIter>
OutputIter setup_src_buf_data(InputIter begin, InputIter end, OutputIter out_buf)
{
  std::for_each(begin, end, [&out_buf](column_view const& col) {
    if (col.nullable()) {
      *out_buf = reinterpret_cast<uint8_t const*>(col.null_mask());
      out_buf++;
    }
    // NOTE: we're always returning the base pointer here.  column-level offset is accounted
    // for later. Also, for some column types (string, list, struct) this pointer will be null
    // because there is no associated data with the root column.
    *out_buf = col.head<uint8_t>();
    out_buf++;

    out_buf = setup_src_buf_data(col.child_begin(), col.child_end(), out_buf);
  });
  return out_buf;
}

/**
 * @brief Count the total number of source buffers we will be copying
 * from.
 *
 * This count includes buffers for all input columns. For example a
 * fixed-width column with validity would be 2 buffers (data, validity).
 * A string column with validity would be 3 buffers (chars, offsets, validity).
 *
 * This function is called recursively in the case of nested types.
 *
 * @param begin Beginning of input columns
 * @param end End of input columns
 *
 * @returns total number of source buffers for this range of columns
 */
template <typename InputIter>
size_type count_src_bufs(InputIter begin, InputIter end)
{
  auto buf_iter = thrust::make_transform_iterator(begin, [](column_view const& col) {
    auto const children_counts = count_src_bufs(col.child_begin(), col.child_end());
    return 1 + (col.nullable() ? 1 : 0) + children_counts;
  });
  return std::accumulate(buf_iter, buf_iter + std::distance(begin, end), 0);
}

/**
 * @brief Computes source buffer information for the copy kernel.
 *
 * For each input column to be split we need to know several pieces of information
 * in the copy kernel.  This function traverses the input columns and prepares this
 * information for the gpu.
 *
 * This function is called recursively in the case of nested types.
 *
 * @param begin Beginning of input columns
 * @param end End of input columns
 * @param head Beginning of source buffer info array
 * @param current Current source buffer info to be written to
 * @param offset_stack_pos Integer representing our current offset nesting depth
 * (how many list or string levels deep we are)
 * @param parent_offset_index Index into src_buf_info output array indicating our nearest
 * containing list parent. -1 if we have no list parent
 * @param offset_depth Current offset nesting depth (how many list levels deep we are)
 *
 * @returns next src_buf_output after processing this range of input columns
 */
// setup source buf info
template <typename InputIter>
std::pair<src_buf_info*, size_type> setup_source_buf_info(InputIter begin,
                                                          InputIter end,
                                                          src_buf_info* head,
                                                          src_buf_info* current,
                                                          rmm::cuda_stream_view stream,
                                                          int offset_stack_pos    = 0,
                                                          int parent_offset_index = -1,
                                                          int offset_depth        = 0);

/**
 * @brief Functor that builds source buffer information based on input columns.
 *
 * Called by setup_source_buf_info to build information for a single source column.  This function
 * will recursively call setup_source_buf_info in the case of nested types.
 */
struct buf_info_functor {
  src_buf_info* head;

  template <typename T>
  std::pair<src_buf_info*, size_type> operator()(column_view const& col,
                                                 src_buf_info* current,
                                                 int offset_stack_pos,
                                                 int parent_offset_index,
                                                 int offset_depth,
                                                 rmm::cuda_stream_view)
  {
    if (col.nullable()) {
      std::tie(current, offset_stack_pos) =
        add_null_buffer(col, current, offset_stack_pos, parent_offset_index, offset_depth);
    }

    // info for the data buffer
    *current = src_buf_info(
      col.type().id(), nullptr, offset_stack_pos, parent_offset_index, false, col.offset());

    return {current + 1, offset_stack_pos + offset_depth};
  }

  template <typename T, typename... Args>
  std::enable_if_t<std::is_same_v<T, cudf::dictionary32>, std::pair<src_buf_info*, size_type>>
  operator()(Args&&...)
  {
    CUDF_FAIL("Unsupported type");
  }

 private:
  std::pair<src_buf_info*, size_type> add_null_buffer(column_view const& col,
                                                      src_buf_info* current,
                                                      int offset_stack_pos,
                                                      int parent_offset_index,
                                                      int offset_depth)
  {
    // info for the validity buffer
    *current = src_buf_info(
      type_id::INT32, nullptr, offset_stack_pos, parent_offset_index, true, col.offset());

    return {current + 1, offset_stack_pos + offset_depth};
  }
};

template <>
std::pair<src_buf_info*, size_type> buf_info_functor::operator()<cudf::string_view>(
  column_view const& col,
  src_buf_info* current,
  int offset_stack_pos,
  int parent_offset_index,
  int offset_depth,
  rmm::cuda_stream_view stream)
{
  if (col.nullable()) {
    std::tie(current, offset_stack_pos) =
      add_null_buffer(col, current, offset_stack_pos, parent_offset_index, offset_depth);
  }

  // the way strings are arranged, the strings column itself contains char data, but our child
  // offsets column actually contains our offsets. So our parent_offset_index is actually our child.

  // string columns don't necessarily have children if they are empty
  auto const has_offsets_child = col.num_children() > 0;

  // string columns contain the underlying chars data.
  *current = src_buf_info(type_id::STRING,
                          nullptr,
                          offset_stack_pos,
                          // if I have an offsets child, it's index will be my parent_offset_index
                          has_offsets_child ? ((current + 1) - head) : parent_offset_index,
                          false,
                          col.offset());

  // if I have offsets, I need to include that in the stack size
  offset_stack_pos += has_offsets_child ? offset_depth + 1 : offset_depth;
  current++;

  if (has_offsets_child) {
    CUDF_EXPECTS(col.num_children() == 1, "Encountered malformed string column");
    strings_column_view scv(col);

    // info for the offsets buffer
    auto offset_col = current;
    CUDF_EXPECTS(not scv.offsets().nullable(), "Encountered nullable string offsets column");
    *current = src_buf_info(type_id::INT32,
                            // note: offsets can be null in the case where the string column
                            // has been created with empty_like().
                            scv.offsets().begin<cudf::id_to_type<type_id::INT32>>(),
                            offset_stack_pos,
                            parent_offset_index,
                            false,
                            col.offset());

    current++;
    offset_stack_pos += offset_depth;

    // since we are crossing an offset boundary, calculate our new depth and parent offset index.
    offset_depth++;
    parent_offset_index = offset_col - head;
  }

  return {current, offset_stack_pos};
}

template <>
std::pair<src_buf_info*, size_type> buf_info_functor::operator()<cudf::list_view>(
  column_view const& col,
  src_buf_info* current,
  int offset_stack_pos,
  int parent_offset_index,
  int offset_depth,
  rmm::cuda_stream_view stream)
{
  lists_column_view lcv(col);

  if (col.nullable()) {
    std::tie(current, offset_stack_pos) =
      add_null_buffer(col, current, offset_stack_pos, parent_offset_index, offset_depth);
  }

  // list columns hold no actual data, but we need to keep a record
  // of it so we know it's size when we are constructing the output columns
  *current = src_buf_info(
    type_id::LIST, nullptr, offset_stack_pos, parent_offset_index, false, col.offset());
  current++;
  offset_stack_pos += offset_depth;

  CUDF_EXPECTS(col.num_children() == 2, "Encountered malformed list column");

  // info for the offsets buffer
  auto offset_col = current;
  *current        = src_buf_info(type_id::INT32,
                          // note: offsets can be null in the case where the lists column
                          // has been created with empty_like().
                          lcv.offsets().begin<cudf::id_to_type<type_id::INT32>>(),
                          offset_stack_pos,
                          parent_offset_index,
                          false,
                          col.offset());
  current++;
  offset_stack_pos += offset_depth;

  // since we are crossing an offset boundary, calculate our new depth and parent offset index.
  offset_depth++;
  parent_offset_index = offset_col - head;

  return setup_source_buf_info(col.child_begin() + 1,
                               col.child_end(),
                               head,
                               current,
                               stream,
                               offset_stack_pos,
                               parent_offset_index,
                               offset_depth);
}

template <>
std::pair<src_buf_info*, size_type> buf_info_functor::operator()<cudf::struct_view>(
  column_view const& col,
  src_buf_info* current,
  int offset_stack_pos,
  int parent_offset_index,
  int offset_depth,
  rmm::cuda_stream_view stream)
{
  if (col.nullable()) {
    std::tie(current, offset_stack_pos) =
      add_null_buffer(col, current, offset_stack_pos, parent_offset_index, offset_depth);
  }

  // struct columns hold no actual data, but we need to keep a record
  // of it so we know it's size when we are constructing the output columns
  *current = src_buf_info(
    type_id::STRUCT, nullptr, offset_stack_pos, parent_offset_index, false, col.offset());
  current++;
  offset_stack_pos += offset_depth;

  // recurse on children
  cudf::structs_column_view scv(col);
  std::vector<column_view> sliced_children;
  sliced_children.reserve(scv.num_children());
  std::transform(
    thrust::make_counting_iterator(0),
    thrust::make_counting_iterator(scv.num_children()),
    std::back_inserter(sliced_children),
    [&scv, &stream](size_type child_index) { return scv.get_sliced_child(child_index, stream); });
  return setup_source_buf_info(sliced_children.begin(),
                               sliced_children.end(),
                               head,
                               current,
                               stream,
                               offset_stack_pos,
                               parent_offset_index,
                               offset_depth);
}

template <typename InputIter>
std::pair<src_buf_info*, size_type> setup_source_buf_info(InputIter begin,
                                                          InputIter end,
                                                          src_buf_info* head,
                                                          src_buf_info* current,
                                                          rmm::cuda_stream_view stream,
                                                          int offset_stack_pos,
                                                          int parent_offset_index,
                                                          int offset_depth)
{
  std::for_each(begin, end, [&](column_view const& col) {
    std::tie(current, offset_stack_pos) = cudf::type_dispatcher(col.type(),
                                                                buf_info_functor{head},
                                                                col,
                                                                current,
                                                                offset_stack_pos,
                                                                parent_offset_index,
                                                                offset_depth,
                                                                stream);
  });
  return {current, offset_stack_pos};
}

/**
 * @brief Given a column, processed split buffers, and a metadata builder, populate
 * the metadata for this column in the builder, and return a tuple of:
 * column size, data offset, bitmask offset and null count.
 *
 * @param src column_view to create metadata from
 * @param current_info dst_buf_info pointer reference, pointing to this column's buffer info
 *                     This is a pointer reference because it is updated by this function as the
 *                     columns's validity and data buffers are visited
 * @param mb A metadata_builder instance to update with the column's packed metadata
 * @param use_src_null_count True for the chunked_pack case where current_info has invalid null
 *                           count information. The null count should be taken
 *                           from `src` because this case is restricted to a single partition
 *                           (no splits)
 * @returns a std::tuple containing:
 *          column size, data offset, bitmask offset, and null count
 */
template <typename BufInfo>
std::tuple<size_type, int64_t, int64_t, size_type> build_output_column_metadata(
  column_view const& src,
  BufInfo& current_info,
  detail::metadata_builder& mb,
  bool use_src_null_count)
{
  auto [bitmask_offset, null_count] = [&]() {
    if (src.nullable()) {
      // offsets in the existing serialized_column metadata are int64_t
      // that's the reason for the casting in this code.
      int64_t const bitmask_offset =
        current_info->num_elements == 0
          ? -1  // this means that the bitmask buffer pointer should be nullptr
          : static_cast<int64_t>(current_info->dst_offset);

      // use_src_null_count is used for the chunked contig split case, where we have
      // no splits: the null_count is just the source column's null_count
      size_type const null_count = use_src_null_count
                                     ? src.null_count()
                                     : (current_info->num_elements == 0
                                          ? 0
                                          : (current_info->num_rows - current_info->valid_count));

      ++current_info;
      return std::pair(bitmask_offset, null_count);
    }
    return std::pair(static_cast<int64_t>(-1), 0);
  }();

  // size/data pointer for the column
  auto const col_size = [&]() {
    // if I am a string column, I need to use the number of rows from my child offset column. the
    // number of rows in my dst_buf_info struct will be equal to the number of chars, which is
    // incorrect. this is a quirk of how cudf stores strings.
    if (src.type().id() == type_id::STRING) {
      // if I have no children (no offsets), then I must have a row count of 0
      if (src.num_children() == 0) { return 0; }

      // otherwise my actual number of rows will be the num_rows field of the next dst_buf_info
      // struct (our child offsets column)
      return (current_info + 1)->num_rows;
    }

    // otherwise the number of rows is the number of elements
    return static_cast<size_type>(current_info->num_elements);
  }();
  int64_t const data_offset =
    col_size == 0 || src.head() == nullptr ? -1 : static_cast<int64_t>(current_info->dst_offset);

  mb.add_column_info_to_meta(
    src.type(), col_size, null_count, data_offset, bitmask_offset, src.num_children());

  ++current_info;
  return {col_size, data_offset, bitmask_offset, null_count};
}

/**
 * @brief Given a set of input columns and processed split buffers, produce
 * output columns.
 *
 * After performing the split we are left with 1 large buffer per incoming split
 * partition.  We need to traverse this buffer and distribute the individual
 * subpieces that represent individual columns and children to produce the final
 * output columns.
 *
 * This function is called recursively in the case of nested types.
 *
 * @param begin Beginning of input columns
 * @param end End of input columns
 * @param info_begin Iterator of dst_buf_info structs containing information about each
 * copied buffer
 * @param out_begin Output iterator of column views
 * @param base_ptr Pointer to the base address of copied data for the working partition
 *
 * @returns new dst_buf_info iterator after processing this range of input columns
 */
template <typename InputIter, typename BufInfo, typename Output>
BufInfo build_output_columns(InputIter begin,
                             InputIter end,
                             BufInfo info_begin,
                             Output out_begin,
                             uint8_t const* const base_ptr,
                             detail::metadata_builder& mb)
{
  auto current_info = info_begin;
  std::transform(begin, end, out_begin, [&current_info, base_ptr, &mb](column_view const& src) {
    auto [col_size, data_offset, bitmask_offset, null_count] =
      build_output_column_metadata<BufInfo>(src, current_info, mb, false);

    auto const bitmask_ptr =
      base_ptr != nullptr && bitmask_offset != -1
        ? reinterpret_cast<bitmask_type const*>(base_ptr + static_cast<uint64_t>(bitmask_offset))
        : nullptr;

    // size/data pointer for the column
    uint8_t const* data_ptr = base_ptr != nullptr && data_offset != -1
                                ? base_ptr + static_cast<uint64_t>(data_offset)
                                : nullptr;

    // children
    auto children = std::vector<column_view>{};
    children.reserve(src.num_children());

    current_info = build_output_columns(
      src.child_begin(), src.child_end(), current_info, std::back_inserter(children), base_ptr, mb);

    return column_view{
      src.type(), col_size, data_ptr, bitmask_ptr, null_count, 0, std::move(children)};
  });

  return current_info;
}

/**
 * @brief Given a set of input columns, processed split buffers, and a metadata_builder,
 * append column metadata using the builder.
 *
 * After performing the split we are left with 1 large buffer per incoming split
 * partition.  We need to traverse this buffer and distribute the individual
 * subpieces that represent individual columns and children to produce the final
 * output columns.
 *
 * This function is called recursively in the case of nested types.
 *
 * @param begin Beginning of input columns
 * @param end End of input columns
 * @param info_begin Iterator of dst_buf_info structs containing information about each
 * copied buffer
 * @param mb packed column metadata builder
 *
 * @returns new dst_buf_info iterator after processing this range of input columns
 */
template <typename InputIter, typename BufInfo>
BufInfo populate_metadata(InputIter begin,
                          InputIter end,
                          BufInfo info_begin,
                          detail::metadata_builder& mb)
{
  auto current_info = info_begin;
  std::for_each(begin, end, [&current_info, &mb](column_view const& src) {
    build_output_column_metadata<BufInfo>(src, current_info, mb, true);

    // children
    current_info = populate_metadata(src.child_begin(), src.child_end(), current_info, mb);
  });

  return current_info;
}

/**
 * @brief Functor that retrieves the size of a destination buffer
 */
struct buf_size_functor {
  dst_buf_info const* ci;
  std::size_t operator() __device__(int index) { return ci[index].buf_size; }
};

/**
 * @brief Functor that retrieves the split "key" for a given output
 * buffer index.
 *
 * The key is simply the partition index.
 */
struct split_key_functor {
  int const num_src_bufs;
  int operator() __device__(int buf_index) const { return buf_index / num_src_bufs; }
};

/**
 * @brief Output iterator for writing values to the dst_offset field of the
 * dst_buf_info struct
 */
struct dst_offset_output_iterator {
  dst_buf_info* c;
  using value_type        = std::size_t;
  using difference_type   = std::size_t;
  using pointer           = std::size_t*;
  using reference         = std::size_t&;
  using iterator_category = thrust::output_device_iterator_tag;

  dst_offset_output_iterator operator+ __host__ __device__(int i) { return {c + i}; }

  dst_offset_output_iterator& operator++ __host__ __device__()
  {
    c++;
    return *this;
  }

  reference operator[] __device__(int i) { return dereference(c + i); }
  reference operator* __device__() { return dereference(c); }

 private:
  reference __device__ dereference(dst_buf_info* c) { return c->dst_offset; }
};

/**
 * @brief Output iterator for writing values to the valid_count field of the
 * dst_buf_info struct
 */
struct dst_valid_count_output_iterator {
  dst_buf_info* c;
  using value_type        = size_type;
  using difference_type   = size_type;
  using pointer           = size_type*;
  using reference         = size_type&;
  using iterator_category = thrust::output_device_iterator_tag;

  dst_valid_count_output_iterator operator+ __host__ __device__(int i) { return {c + i}; }

  dst_valid_count_output_iterator& operator++ __host__ __device__()
  {
    c++;
    return *this;
  }

  reference operator[] __device__(int i) { return dereference(c + i); }
  reference operator* __device__() { return dereference(c); }

 private:
  reference __device__ dereference(dst_buf_info* c) { return c->valid_count; }
};

/**
 * @brief Functor for computing size of data elements for a given cudf type.
 *
 * Note: columns types which themselves inherently have no data (strings, lists,
 * structs) return 0.
 */
struct size_of_helper {
  template <typename T>
  constexpr std::enable_if_t<!is_fixed_width<T>() && !std::is_same_v<T, cudf::string_view>, int>
    __device__ operator()() const
  {
    return 0;
  }

  template <typename T>
  constexpr std::enable_if_t<!is_fixed_width<T>() && std::is_same_v<T, cudf::string_view>, int>
    __device__ operator()() const
  {
    return sizeof(cudf::device_storage_type_t<int8_t>);
  }

  template <typename T>
  constexpr std::enable_if_t<is_fixed_width<T>(), int> __device__ operator()() const noexcept
  {
    return sizeof(cudf::device_storage_type_t<T>);
  }
};

/**
 * @brief Functor for returning the number of batches an input buffer is being
 * subdivided into during the repartitioning step.
 *
 * Note: columns types which themselves inherently have no data (strings, lists,
 * structs) return 0.
 */
struct num_batches_func {
  thrust::pair<std::size_t, std::size_t> const* const batches;
  __device__ std::size_t operator()(size_type i) const { return thrust::get<0>(batches[i]); }
};

/**
 * @brief Get the size in bytes of a batch described by `dst_buf_info`.
 */
struct batch_byte_size_function {
  size_type const num_batches;
  dst_buf_info const* const infos;
  __device__ std::size_t operator()(size_type i) const
  {
    if (i == num_batches) { return 0; }
    auto const& buf = *(infos + i);
    std::size_t const bytes =
      static_cast<std::size_t>(buf.num_elements) * static_cast<std::size_t>(buf.element_size);
    return util::round_up_unsafe(bytes, split_align);
  }
};

/**
 * @brief Get the input buffer index given the output buffer index.
 */
struct out_to_in_index_function {
  size_type const* const batch_offsets;
  int const num_bufs;
  __device__ int operator()(size_type i) const
  {
    return static_cast<size_type>(
             thrust::upper_bound(thrust::seq, batch_offsets, batch_offsets + num_bufs + 1, i) -
             batch_offsets) -
           1;
  }
};

// packed block of memory 1: split indices and src_buf_info structs
struct packed_split_indices_and_src_buf_info {
  packed_split_indices_and_src_buf_info(cudf::table_view const& input,
                                        std::vector<size_type> const& splits,
                                        std::size_t num_partitions,
                                        cudf::size_type num_src_bufs,
                                        rmm::cuda_stream_view stream,
                                        rmm::mr::device_memory_resource* temp_mr)
    : indices_size(
        cudf::util::round_up_safe((num_partitions + 1) * sizeof(size_type), split_align)),
      src_buf_info_size(
        cudf::util::round_up_safe(num_src_bufs * sizeof(src_buf_info), split_align)),
      // host-side
      h_indices_and_source_info(indices_size + src_buf_info_size),
      h_indices{reinterpret_cast<size_type*>(h_indices_and_source_info.data())},
      h_src_buf_info{
        reinterpret_cast<src_buf_info*>(h_indices_and_source_info.data() + indices_size)}
  {
    // compute splits -> indices.
    // these are row numbers per split
    h_indices[0]              = 0;
    h_indices[num_partitions] = input.column(0).size();
    std::copy(splits.begin(), splits.end(), std::next(h_indices));

    // setup source buf info
    setup_source_buf_info(input.begin(), input.end(), h_src_buf_info, h_src_buf_info, stream);

    offset_stack_partition_size = compute_offset_stack_size(input.begin(), input.end());
    offset_stack_size           = offset_stack_partition_size * num_partitions * sizeof(size_type);
    // device-side
    // gpu-only : stack space needed for nested list offset calculation
    d_indices_and_source_info =
      rmm::device_buffer(indices_size + src_buf_info_size + offset_stack_size, stream, temp_mr);
    d_indices      = reinterpret_cast<size_type*>(d_indices_and_source_info.data());
    d_src_buf_info = reinterpret_cast<src_buf_info*>(
      reinterpret_cast<uint8_t*>(d_indices_and_source_info.data()) + indices_size);
    d_offset_stack =
      reinterpret_cast<size_type*>(reinterpret_cast<uint8_t*>(d_indices_and_source_info.data()) +
                                   indices_size + src_buf_info_size);

    CUDF_CUDA_TRY(hipMemcpyAsync(
      d_indices, h_indices, indices_size + src_buf_info_size, hipMemcpyDefault, stream.value()));
  }

  size_type const indices_size;
  std::size_t const src_buf_info_size;
  std::size_t offset_stack_size;

  std::vector<uint8_t> h_indices_and_source_info;
  rmm::device_buffer d_indices_and_source_info;

  size_type* const h_indices;
  src_buf_info* const h_src_buf_info;

  int offset_stack_partition_size;
  size_type* d_indices;
  src_buf_info* d_src_buf_info;
  size_type* d_offset_stack;
};

// packed block of memory 2: partition buffer sizes and dst_buf_info structs
struct packed_partition_buf_size_and_dst_buf_info {
  packed_partition_buf_size_and_dst_buf_info(std::size_t num_partitions,
                                             std::size_t num_bufs,
                                             rmm::cuda_stream_view stream,
                                             rmm::mr::device_memory_resource* temp_mr)
    : stream(stream),
      buf_sizes_size{cudf::util::round_up_safe(num_partitions * sizeof(std::size_t), split_align)},
      dst_buf_info_size{cudf::util::round_up_safe(num_bufs * sizeof(dst_buf_info), split_align)},
      // host-side
      h_buf_sizes_and_dst_info(buf_sizes_size + dst_buf_info_size),
      h_buf_sizes{reinterpret_cast<std::size_t*>(h_buf_sizes_and_dst_info.data())},
      h_dst_buf_info{
        reinterpret_cast<dst_buf_info*>(h_buf_sizes_and_dst_info.data() + buf_sizes_size)},
      // device-side
      d_buf_sizes_and_dst_info(buf_sizes_size + dst_buf_info_size, stream, temp_mr),
      d_buf_sizes{reinterpret_cast<std::size_t*>(d_buf_sizes_and_dst_info.data())},
      // destination buffer info
      d_dst_buf_info{reinterpret_cast<dst_buf_info*>(
        static_cast<uint8_t*>(d_buf_sizes_and_dst_info.data()) + buf_sizes_size)}
  {
  }

  void copy_to_host()
  {
    // DtoH buf sizes and col info back to the host
    CUDF_CUDA_TRY(hipMemcpyAsync(h_buf_sizes,
                                  d_buf_sizes,
                                  buf_sizes_size + dst_buf_info_size,
                                  hipMemcpyDefault,
                                  stream.value()));
  }

  rmm::cuda_stream_view const stream;

  // buffer sizes and destination info (used in batched copies)
  std::size_t const buf_sizes_size;
  std::size_t const dst_buf_info_size;

  std::vector<uint8_t> h_buf_sizes_and_dst_info;
  std::size_t* const h_buf_sizes;
  dst_buf_info* const h_dst_buf_info;

  rmm::device_buffer d_buf_sizes_and_dst_info;
  std::size_t* const d_buf_sizes;
  dst_buf_info* const d_dst_buf_info;
};

// Packed block of memory 3:
// Pointers to source and destination buffers (and stack space on the
// gpu for offset computation)
struct packed_src_and_dst_pointers {
  packed_src_and_dst_pointers(cudf::table_view const& input,
                              std::size_t num_partitions,
                              cudf::size_type num_src_bufs,
                              rmm::cuda_stream_view stream,
                              rmm::mr::device_memory_resource* temp_mr)
    : stream(stream),
      src_bufs_size{cudf::util::round_up_safe(num_src_bufs * sizeof(uint8_t*), split_align)},
      dst_bufs_size{cudf::util::round_up_safe(num_partitions * sizeof(uint8_t*), split_align)},
      // host-side
      h_src_and_dst_buffers(src_bufs_size + dst_bufs_size),
      h_src_bufs{reinterpret_cast<uint8_t const**>(h_src_and_dst_buffers.data())},
      h_dst_bufs{reinterpret_cast<uint8_t**>(h_src_and_dst_buffers.data() + src_bufs_size)},
      // device-side
      d_src_and_dst_buffers{rmm::device_buffer(src_bufs_size + dst_bufs_size, stream, temp_mr)},
      d_src_bufs{reinterpret_cast<uint8_t const**>(d_src_and_dst_buffers.data())},
      d_dst_bufs{reinterpret_cast<uint8_t**>(
        reinterpret_cast<uint8_t*>(d_src_and_dst_buffers.data()) + src_bufs_size)}
  {
    // setup src buffers
    setup_src_buf_data(input.begin(), input.end(), h_src_bufs);
  }

  void copy_to_device()
  {
    CUDF_CUDA_TRY(hipMemcpyAsync(d_src_and_dst_buffers.data(),
                                  h_src_and_dst_buffers.data(),
                                  src_bufs_size + dst_bufs_size,
                                  hipMemcpyDefault,
                                  stream.value()));
  }

  rmm::cuda_stream_view const stream;
  std::size_t const src_bufs_size;
  std::size_t const dst_bufs_size;

  std::vector<uint8_t> h_src_and_dst_buffers;
  uint8_t const** const h_src_bufs;
  uint8_t** const h_dst_bufs;

  rmm::device_buffer d_src_and_dst_buffers;
  uint8_t const** const d_src_bufs;
  uint8_t** const d_dst_bufs;
};

/**
 * @brief Create an instance of `packed_src_and_dst_pointers` populating destination
 * partitition buffers (if any) from `out_buffers`. In the chunked_pack case
 * `out_buffers` is empty, and the destination pointer is provided separately
 * to the `copy_partitions` kernel.
 *
 * @param input source table view
 * @param num_partitions the number of partitions (1 meaning no splits)
 * @param num_src_bufs number of buffers for the source columns including children
 * @param out_buffers the destination buffers per partition if in the non-chunked case
 * @param stream Optional CUDA stream on which to execute kernels
 * @param temp_mr A memory resource for temporary and scratch space
 *
 * @returns new unique pointer to packed_src_and_dst_pointers
 */
std::unique_ptr<packed_src_and_dst_pointers> setup_src_and_dst_pointers(
  cudf::table_view const& input,
  std::size_t num_partitions,
  cudf::size_type num_src_bufs,
  std::vector<rmm::device_buffer>& out_buffers,
  rmm::cuda_stream_view stream,
  rmm::mr::device_memory_resource* temp_mr)
{
  auto src_and_dst_pointers = std::make_unique<packed_src_and_dst_pointers>(
    input, num_partitions, num_src_bufs, stream, temp_mr);

  std::transform(
    out_buffers.begin(), out_buffers.end(), src_and_dst_pointers->h_dst_bufs, [](auto& buf) {
      return static_cast<uint8_t*>(buf.data());
    });

  // copy the struct to device memory to access from the kernel
  src_and_dst_pointers->copy_to_device();

  return src_and_dst_pointers;
}

/**
 * @brief Create an instance of `packed_partition_buf_size_and_dst_buf_info` containing
 * the partition-level dst_buf_info structs for each partition and column buffer.
 *
 * @param input source table view
 * @param splits the numeric value (in rows) for each split, empty for 1 partition
 * @param num_partitions the number of partitions create (1 meaning no splits)
 * @param num_src_bufs number of buffers for the source columns including children
 * @param num_bufs num_src_bufs times the number of partitions
 * @param stream Optional CUDA stream on which to execute kernels
 * @param temp_mr A memory resource for temporary and scratch space
 *
 * @returns new unique pointer to `packed_partition_buf_size_and_dst_buf_info`
 */
std::unique_ptr<packed_partition_buf_size_and_dst_buf_info> compute_splits(
  cudf::table_view const& input,
  std::vector<size_type> const& splits,
  std::size_t num_partitions,
  cudf::size_type num_src_bufs,
  std::size_t num_bufs,
  rmm::cuda_stream_view stream,
  rmm::mr::device_memory_resource* temp_mr)
{
  auto partition_buf_size_and_dst_buf_info =
    std::make_unique<packed_partition_buf_size_and_dst_buf_info>(
      num_partitions, num_bufs, stream, temp_mr);

  auto const d_dst_buf_info = partition_buf_size_and_dst_buf_info->d_dst_buf_info;
  auto const d_buf_sizes    = partition_buf_size_and_dst_buf_info->d_buf_sizes;

  auto const split_indices_and_src_buf_info = packed_split_indices_and_src_buf_info(
    input, splits, num_partitions, num_src_bufs, stream, temp_mr);

  auto const d_src_buf_info = split_indices_and_src_buf_info.d_src_buf_info;
  auto const offset_stack_partition_size =
    split_indices_and_src_buf_info.offset_stack_partition_size;
  auto const d_offset_stack = split_indices_and_src_buf_info.d_offset_stack;
  auto const d_indices      = split_indices_and_src_buf_info.d_indices;

  // compute sizes of each column in each partition, including alignment.
  thrust::transform(
    rmm::exec_policy(stream, temp_mr),
    thrust::make_counting_iterator<std::size_t>(0),
    thrust::make_counting_iterator<std::size_t>(num_bufs),
    d_dst_buf_info,
    cuda::proclaim_return_type<dst_buf_info>([d_src_buf_info,
                                              offset_stack_partition_size,
                                              d_offset_stack,
                                              d_indices,
                                              num_src_bufs] __device__(std::size_t t) {
      int const split_index   = t / num_src_bufs;
      int const src_buf_index = t % num_src_bufs;
      auto const& src_info    = d_src_buf_info[src_buf_index];

      // apply nested offsets (lists and string columns).
      //
      // We can't just use the incoming row indices to figure out where to read from in a
      // nested list situation.  We have to apply offsets every time we cross a boundary
      // (list or string).  This loop applies those offsets so that our incoming row_index_start
      // and row_index_end get transformed to our final values.
      //
      int const stack_pos = src_info.offset_stack_pos + (split_index * offset_stack_partition_size);
      size_type* offset_stack  = &d_offset_stack[stack_pos];
      int parent_offsets_index = src_info.parent_offsets_index;
      int stack_size           = 0;
      int root_column_offset   = src_info.column_offset;
      while (parent_offsets_index >= 0) {
        offset_stack[stack_size++] = parent_offsets_index;
        root_column_offset         = d_src_buf_info[parent_offsets_index].column_offset;
        parent_offsets_index       = d_src_buf_info[parent_offsets_index].parent_offsets_index;
      }
      // make sure to include the -column- offset on the root column in our calculation.
      int row_start = d_indices[split_index] + root_column_offset;
      int row_end   = d_indices[split_index + 1] + root_column_offset;
      while (stack_size > 0) {
        stack_size--;
        auto const offsets = d_src_buf_info[offset_stack[stack_size]].offsets;
        // this case can happen when you have empty string or list columns constructed with
        // empty_like()
        if (offsets != nullptr) {
          row_start = offsets[row_start];
          row_end   = offsets[row_end];
        }
      }

      // final element indices and row count
      int const src_element_index = src_info.is_validity ? row_start / 32 : row_start;
      int const num_rows          = row_end - row_start;
      // if I am an offsets column, all my values need to be shifted
      int const value_shift = src_info.offsets == nullptr ? 0 : src_info.offsets[row_start];
      // if I am a validity column, we may need to shift bits
      int const bit_shift = src_info.is_validity ? row_start % 32 : 0;
      // # of rows isn't necessarily the same as # of elements to be copied.
      auto const num_elements = [&]() {
        if (src_info.offsets != nullptr && num_rows > 0) {
          return num_rows + 1;
        } else if (src_info.is_validity) {
          return (num_rows + 31) / 32;
        }
        return num_rows;
      }();
      int const element_size = cudf::type_dispatcher(data_type{src_info.type}, size_of_helper{});
      std::size_t const bytes =
        static_cast<std::size_t>(num_elements) * static_cast<std::size_t>(element_size);

      return dst_buf_info{util::round_up_unsafe(bytes, split_align),
                          num_elements,
                          element_size,
                          num_rows,
                          src_element_index,
                          0,
                          value_shift,
                          bit_shift,
                          src_info.is_validity ? 1 : 0,
                          src_buf_index,
                          split_index};
    }));

  // compute total size of each partition
  // key is the split index
  {
    auto const keys = cudf::detail::make_counting_transform_iterator(
      0, split_key_functor{static_cast<int>(num_src_bufs)});
    auto values =
      cudf::detail::make_counting_transform_iterator(0, buf_size_functor{d_dst_buf_info});

    thrust::reduce_by_key(rmm::exec_policy(stream, temp_mr),
                          keys,
                          keys + num_bufs,
                          values,
                          thrust::make_discard_iterator(),
                          d_buf_sizes);
  }

  // compute start offset for each output buffer for each split
  {
    auto const keys = cudf::detail::make_counting_transform_iterator(
      0, split_key_functor{static_cast<int>(num_src_bufs)});
    auto values =
      cudf::detail::make_counting_transform_iterator(0, buf_size_functor{d_dst_buf_info});

    thrust::exclusive_scan_by_key(rmm::exec_policy(stream, temp_mr),
                                  keys,
                                  keys + num_bufs,
                                  values,
                                  dst_offset_output_iterator{d_dst_buf_info},
                                  std::size_t{0});
  }

  partition_buf_size_and_dst_buf_info->copy_to_host();

  stream.synchronize();

  return partition_buf_size_and_dst_buf_info;
}

/**
 * @brief Struct containing information about the actual batches we will send to the
 * `copy_partitions` kernel and the number of iterations we need to carry out this copy.
 *
 * For the non-chunked contiguous_split case, this contains the batched dst_buf_infos and the
 * number of iterations is going to be 1 since the non-chunked case is single pass.
 *
 * For the chunked_pack case, this also contains the batched dst_buf_infos for all
 * iterations in addition to helping keep the state about what batches have been copied so far
 * and what are the sizes (in bytes) of each iteration.
 */
struct chunk_iteration_state {
  chunk_iteration_state(rmm::device_uvector<dst_buf_info> _d_batched_dst_buf_info,
                        rmm::device_uvector<size_type> _d_batch_offsets,
                        std::vector<std::size_t>&& _h_num_buffs_per_iteration,
                        std::vector<std::size_t>&& _h_size_of_buffs_per_iteration,
                        std::size_t total_size)
    : num_iterations(_h_num_buffs_per_iteration.size()),
      current_iteration{0},
      starting_batch{0},
      d_batched_dst_buf_info(std::move(_d_batched_dst_buf_info)),
      d_batch_offsets(std::move(_d_batch_offsets)),
      h_num_buffs_per_iteration(std::move(_h_num_buffs_per_iteration)),
      h_size_of_buffs_per_iteration(std::move(_h_size_of_buffs_per_iteration)),
      total_size(total_size)
  {
  }

  static std::unique_ptr<chunk_iteration_state> create(
    rmm::device_uvector<thrust::pair<std::size_t, std::size_t>> const& batches,
    int num_bufs,
    dst_buf_info* d_orig_dst_buf_info,
    std::size_t const* const h_buf_sizes,
    std::size_t num_partitions,
    std::size_t user_buffer_size,
    rmm::cuda_stream_view stream,
    rmm::mr::device_memory_resource* temp_mr);

  /**
   * @brief As of the time of the call, return the starting 1MB batch index, and the
   * number of batches to copy.
   *
   * @return the current iteration's starting_batch and batch count as a pair
   */
  std::pair<std::size_t, std::size_t> get_current_starting_index_and_buff_count() const
  {
    CUDF_EXPECTS(current_iteration < num_iterations,
                 "current_iteration cannot exceed num_iterations");
    auto count_for_current = h_num_buffs_per_iteration[current_iteration];
    return {starting_batch, count_for_current};
  }

  /**
   * @brief Advance the iteration state if there are iterations left, updating the
   * starting batch and returning the amount of bytes were copied in the iteration
   * we just finished.
   * @throws cudf::logic_error If the state was at the last iteration before entering
   * this function.
   * @return size in bytes that were copied in the finished iteration
   */
  std::size_t advance_iteration()
  {
    CUDF_EXPECTS(current_iteration < num_iterations,
                 "current_iteration cannot exceed num_iterations");
    std::size_t bytes_copied = h_size_of_buffs_per_iteration[current_iteration];
    starting_batch += h_num_buffs_per_iteration[current_iteration];
    ++current_iteration;
    return bytes_copied;
  }

  /**
   * Returns true if there are iterations left.
   */
  bool has_more_copies() const { return current_iteration < num_iterations; }

  rmm::device_uvector<dst_buf_info> d_batched_dst_buf_info;  ///< dst_buf_info per 1MB batch
  rmm::device_uvector<size_type> const d_batch_offsets;  ///< Offset within a batch per dst_buf_info
  std::size_t const total_size;                          ///< The aggregate size of all iterations
  int const num_iterations;                              ///< The total number of iterations
  int current_iteration;  ///< Marks the current iteration being worked on

 private:
  std::size_t starting_batch;  ///< Starting batch index for the current iteration
  std::vector<std::size_t> const h_num_buffs_per_iteration;  ///< The count of batches per iteration
  std::vector<std::size_t> const
    h_size_of_buffs_per_iteration;  ///< The size in bytes per iteration
};

std::unique_ptr<chunk_iteration_state> chunk_iteration_state::create(
  rmm::device_uvector<thrust::pair<std::size_t, std::size_t>> const& batches,
  int num_bufs,
  dst_buf_info* d_orig_dst_buf_info,
  std::size_t const* const h_buf_sizes,
  std::size_t num_partitions,
  std::size_t user_buffer_size,
  rmm::cuda_stream_view stream,
  rmm::mr::device_memory_resource* temp_mr)
{
  rmm::device_uvector<size_type> d_batch_offsets(num_bufs + 1, stream, temp_mr);

  auto const buf_count_iter = cudf::detail::make_counting_transform_iterator(
    0,
    cuda::proclaim_return_type<std::size_t>(
      [num_bufs, num_batches = num_batches_func{batches.begin()}] __device__(size_type i) {
        return i == num_bufs ? 0 : num_batches(i);
      }));

  thrust::exclusive_scan(rmm::exec_policy(stream, temp_mr),
                         buf_count_iter,
                         buf_count_iter + num_bufs + 1,
                         d_batch_offsets.begin(),
                         0);

  auto const num_batches_iter =
    cudf::detail::make_counting_transform_iterator(0, num_batches_func{batches.begin()});
  size_type const num_batches = thrust::reduce(
    rmm::exec_policy(stream, temp_mr), num_batches_iter, num_batches_iter + batches.size());

  auto out_to_in_index = out_to_in_index_function{d_batch_offsets.begin(), num_bufs};

  auto const iter = thrust::make_counting_iterator(0);

  // load up the batches as d_dst_buf_info
  rmm::device_uvector<dst_buf_info> d_batched_dst_buf_info(num_batches, stream, temp_mr);

  thrust::for_each(
    rmm::exec_policy(stream, temp_mr),
    iter,
    iter + num_batches,
    [d_orig_dst_buf_info,
     d_batched_dst_buf_info = d_batched_dst_buf_info.begin(),
     batches                = batches.begin(),
     d_batch_offsets        = d_batch_offsets.begin(),
     out_to_in_index] __device__(size_type i) {
      size_type const in_buf_index = out_to_in_index(i);
      size_type const batch_index  = i - d_batch_offsets[in_buf_index];
      auto const batch_size        = thrust::get<1>(batches[in_buf_index]);
      dst_buf_info const& in       = d_orig_dst_buf_info[in_buf_index];

      // adjust info
      dst_buf_info& out = d_batched_dst_buf_info[i];
      out.element_size  = in.element_size;
      out.value_shift   = in.value_shift;
      out.bit_shift     = in.bit_shift;
      out.valid_count =
        in.valid_count;  // valid count will be set to 1 if this is a validity buffer
      out.src_buf_index = in.src_buf_index;
      out.dst_buf_index = in.dst_buf_index;

      size_type const elements_per_batch =
        out.element_size == 0 ? 0 : batch_size / out.element_size;
      out.num_elements = ((batch_index + 1) * elements_per_batch) > in.num_elements
                           ? in.num_elements - (batch_index * elements_per_batch)
                           : elements_per_batch;

      size_type const rows_per_batch =
        // if this is a validity buffer, each element is a bitmask_type, which
        // corresponds to 32 rows.
        out.valid_count > 0
          ? elements_per_batch * static_cast<size_type>(cudf::detail::size_in_bits<bitmask_type>())
          : elements_per_batch;
      out.num_rows = ((batch_index + 1) * rows_per_batch) > in.num_rows
                       ? in.num_rows - (batch_index * rows_per_batch)
                       : rows_per_batch;

      out.src_element_index = in.src_element_index + (batch_index * elements_per_batch);
      out.dst_offset        = in.dst_offset + (batch_index * batch_size);

      // out.bytes and out.buf_size are unneeded here because they are only used to
      // calculate real output buffer sizes. the data we are generating here is
      // purely intermediate for the purposes of doing more uniform copying of data
      // underneath the final structure of the output
    });

  /**
   * In the chunked case, this is the code that fixes up the offsets of each batch
   * and prepares each iteration. Given the batches computed before, it figures
   * out the number of batches that will fit in an iteration of `user_buffer_size`.
   *
   * Specifically, offsets for batches are reset to the 0th byte when a new iteration
   * of `user_buffer_size` bytes is needed.
   */
  if (user_buffer_size != 0) {
    // copy the batch offsets back to host
    std::vector<std::size_t> h_offsets(num_batches + 1);
    {
      rmm::device_uvector<std::size_t> offsets(h_offsets.size(), stream, temp_mr);
      auto const batch_byte_size_iter = cudf::detail::make_counting_transform_iterator(
        0, batch_byte_size_function{num_batches, d_batched_dst_buf_info.begin()});

      thrust::exclusive_scan(rmm::exec_policy(stream, temp_mr),
                             batch_byte_size_iter,
                             batch_byte_size_iter + num_batches + 1,
                             offsets.begin());

      CUDF_CUDA_TRY(hipMemcpyAsync(h_offsets.data(),
                                    offsets.data(),
                                    sizeof(std::size_t) * offsets.size(),
                                    hipMemcpyDefault,
                                    stream.value()));

      // the next part is working on the CPU, so we want to synchronize here
      stream.synchronize();
    }

    std::vector<std::size_t> num_batches_per_iteration;
    std::vector<std::size_t> size_of_batches_per_iteration;
    std::vector<std::size_t> accum_size_per_iteration;
    std::size_t accum_size = 0;
    {
      auto current_offset_it = h_offsets.begin();
      // figure out how many iterations we need, while fitting batches to iterations
      // with no more than user_buffer_size bytes worth of batches
      while (current_offset_it != h_offsets.end()) {
        // next_iteration_it points to the batch right above the boundary (the batch
        // that didn't fit).
        auto next_iteration_it =
          std::lower_bound(current_offset_it,
                           h_offsets.end(),
                           // We add the cumulative size + 1 because we want to find what would fit
                           // within a buffer of user_buffer_size (up to user_buffer_size).
                           // Since h_offsets is a prefix scan, we add the size we accumulated so
                           // far so we are looking for the next user_buffer_sized boundary.
                           user_buffer_size + accum_size + 1);

        // we subtract 1 from the number of batch here because next_iteration_it points
        // to the batch that didn't fit, so it's one off.
        auto batches_in_iter = std::distance(current_offset_it, next_iteration_it) - 1;

        // to get the amount of bytes in this iteration we get the prefix scan size
        // and subtract the cumulative size so far, leaving the bytes belonging to this
        // iteration
        auto iter_size_bytes = *(current_offset_it + batches_in_iter) - accum_size;
        accum_size += iter_size_bytes;

        num_batches_per_iteration.push_back(batches_in_iter);
        size_of_batches_per_iteration.push_back(iter_size_bytes);
        accum_size_per_iteration.push_back(accum_size);

        if (next_iteration_it == h_offsets.end()) { break; }

        current_offset_it += batches_in_iter;
      }
    }

    // apply changed offset
    {
      auto d_accum_size_per_iteration =
        cudf::detail::make_device_uvector_async(accum_size_per_iteration, stream, temp_mr);

      // we want to update the offset of batches for every iteration, except the first one (because
      // offsets in the first iteration are all 0 based)
      auto num_batches_in_first_iteration = num_batches_per_iteration[0];
      auto const iter     = thrust::make_counting_iterator(num_batches_in_first_iteration);
      auto num_iterations = accum_size_per_iteration.size();
      thrust::for_each(
        rmm::exec_policy(stream, temp_mr),
        iter,
        iter + num_batches - num_batches_in_first_iteration,
        [num_iterations,
         d_batched_dst_buf_info     = d_batched_dst_buf_info.begin(),
         d_accum_size_per_iteration = d_accum_size_per_iteration.begin()] __device__(size_type i) {
          auto prior_iteration_size =
            thrust::upper_bound(thrust::seq,
                                d_accum_size_per_iteration,
                                d_accum_size_per_iteration + num_iterations,
                                d_batched_dst_buf_info[i].dst_offset) -
            1;
          d_batched_dst_buf_info[i].dst_offset -= *prior_iteration_size;
        });
    }
    return std::make_unique<chunk_iteration_state>(std::move(d_batched_dst_buf_info),
                                                   std::move(d_batch_offsets),
                                                   std::move(num_batches_per_iteration),
                                                   std::move(size_of_batches_per_iteration),
                                                   accum_size);

  } else {
    // we instantiate an "iteration state" for the regular single pass contiguous_split
    // consisting of 1 iteration with all of the batches and totalling `total_size` bytes.
    auto const total_size = std::reduce(h_buf_sizes, h_buf_sizes + num_partitions);

    // 1 iteration with the whole size
    return std::make_unique<chunk_iteration_state>(
      std::move(d_batched_dst_buf_info),
      std::move(d_batch_offsets),
      std::move(std::vector<std::size_t>{static_cast<std::size_t>(num_batches)}),
      std::move(std::vector<std::size_t>{total_size}),
      total_size);
  }
}

/**
 * @brief Create an instance of `chunk_iteration_state` containing 1MB batches of work
 * that are further grouped into chunks or iterations.
 *
 * This function handles both the `chunked_pack` case: when `user_buffer_size` is non-zero,
 * and the single-shot `contiguous_split` case.
 *
 * @param num_bufs num_src_bufs times the number of partitions
 * @param d_dst_buf_info dst_buf_info per partition produced in `compute_splits`
 * @param h_buf_sizes size in bytes of a partition (accessible from host)
 * @param num_partitions the number of partitions (1 meaning no splits)
 * @param user_buffer_size if non-zero, it is the size in bytes that 1MB batches should be
 *        grouped in, as different iterations.
 * @param stream Optional CUDA stream on which to execute kernels
 * @param temp_mr A memory resource for temporary and scratch space
 *
 * @returns new unique pointer to `chunk_iteration_state`
 */
std::unique_ptr<chunk_iteration_state> compute_batches(int num_bufs,
                                                       dst_buf_info* const d_dst_buf_info,
                                                       std::size_t const* const h_buf_sizes,
                                                       std::size_t num_partitions,
                                                       std::size_t user_buffer_size,
                                                       rmm::cuda_stream_view stream,
                                                       rmm::mr::device_memory_resource* temp_mr)
{
  // Since we parallelize at one block per copy, performance is vulnerable to situations where we
  // have small numbers of copies to do (a combination of small numbers of splits and/or columns),
  // so we will take the actual set of outgoing source/destination buffers and further partition
  // them into much smaller batches in order to drive up the number of blocks and overall
  // occupancy.
  rmm::device_uvector<thrust::pair<std::size_t, std::size_t>> batches(num_bufs, stream, temp_mr);
  thrust::transform(
    rmm::exec_policy(stream, temp_mr),
    d_dst_buf_info,
    d_dst_buf_info + num_bufs,
    batches.begin(),
    cuda::proclaim_return_type<thrust::pair<std::size_t, std::size_t>>(
      [desired_batch_size = desired_batch_size] __device__(
        dst_buf_info const& buf) -> thrust::pair<std::size_t, std::size_t> {
        // Total bytes for this incoming partition
        std::size_t const bytes =
          static_cast<std::size_t>(buf.num_elements) * static_cast<std::size_t>(buf.element_size);

        // This clause handles nested data types (e.g. list or string) that store no data in the row
        // columns, only in their children.
        if (bytes == 0) { return {1, 0}; }

        // The number of batches we want to subdivide this buffer into
        std::size_t const num_batches = std::max(
          std::size_t{1}, util::round_up_unsafe(bytes, desired_batch_size) / desired_batch_size);

        // NOTE: leaving batch size as a separate parameter for future tuning
        // possibilities, even though in the current implementation it will be a
        // constant.
        return {num_batches, desired_batch_size};
      }));

  return chunk_iteration_state::create(batches,
                                       num_bufs,
                                       d_dst_buf_info,
                                       h_buf_sizes,
                                       num_partitions,
                                       user_buffer_size,
                                       stream,
                                       temp_mr);
}

void copy_data(int num_batches_to_copy,
               int starting_batch,
               uint8_t const** d_src_bufs,
               uint8_t** d_dst_bufs,
               rmm::device_uvector<dst_buf_info>& d_dst_buf_info,
               uint8_t* user_buffer,
               rmm::cuda_stream_view stream)
{
  constexpr size_type block_size = 256;
  if (user_buffer != nullptr) {
    auto index_to_buffer = [user_buffer] __device__(unsigned int) { return user_buffer; };
    copy_partitions<block_size><<<num_batches_to_copy, block_size, 0, stream.value()>>>(
      index_to_buffer, d_src_bufs, d_dst_buf_info.data() + starting_batch);
  } else {
    auto index_to_buffer = [d_dst_bufs,
                            dst_buf_info = d_dst_buf_info.data(),
                            user_buffer] __device__(unsigned int buf_index) {
      auto const dst_buf_index = dst_buf_info[buf_index].dst_buf_index;
      return d_dst_bufs[dst_buf_index];
    };
    copy_partitions<block_size><<<num_batches_to_copy, block_size, 0, stream.value()>>>(
      index_to_buffer, d_src_bufs, d_dst_buf_info.data() + starting_batch);
  }
}

/**
 * @brief Function that checks an input table_view and splits for specific edge cases.
 *
 * It will return true if the input is "empty" (no rows or columns), which means
 * special handling has to happen in the calling code.
 *
 * @param input table_view of source table to be split
 * @param splits the splits specified by the user, or an empty vector if no splits
 * @returns true if the input is empty, false otherwise
 */
bool check_inputs(cudf::table_view const& input, std::vector<size_type> const& splits)
{
  if (input.num_columns() == 0) { return true; }
  if (splits.size() > 0) {
    CUDF_EXPECTS(splits.back() <= input.column(0).size(),
                 "splits can't exceed size of input columns",
                 std::out_of_range);
  }
  size_type begin = 0;
  for (auto end : splits) {
    CUDF_EXPECTS(begin >= 0, "Starting index cannot be negative.", std::out_of_range);
    CUDF_EXPECTS(
      end >= begin, "End index cannot be smaller than the starting index.", std::invalid_argument);
    CUDF_EXPECTS(end <= input.column(0).size(), "Slice range out of bounds.", std::out_of_range);
    begin = end;
  }
  return input.column(0).size() == 0;
}

};  // anonymous namespace

namespace detail {

/**
 * @brief A helper struct containing the state of contiguous_split, whether the caller
 * is using the single-pass contiguous_split or chunked_pack.
 *
 * It exposes an iterator-like pattern where contiguous_split_state::has_next()
 * returns true when there is work to be done, and false otherwise.
 *
 * contiguous_split_state::contiguous_split() performs a single-pass contiguous_split
 * and is valid iff contiguous_split_state is instantiated with 0 for the user_buffer_size.
 *
 * contiguous_split_state::contiguous_split_chunk(device_span) is only valid when
 * user_buffer_size > 0. It should be called as long as has_next() returns true. The
 * device_span passed to contiguous_split_chunk must be allocated in stream `stream` by
 * the user.
 *
 * None of the methods are thread safe.
 */
struct contiguous_split_state {
  contiguous_split_state(cudf::table_view const& input,
                         std::size_t user_buffer_size,
                         rmm::cuda_stream_view stream,
                         rmm::mr::device_memory_resource* mr,
                         rmm::mr::device_memory_resource* temp_mr)
    : contiguous_split_state(input, {}, user_buffer_size, stream, mr, temp_mr)
  {
  }

  contiguous_split_state(cudf::table_view const& input,
                         std::vector<size_type> const& splits,
                         rmm::cuda_stream_view stream,
                         rmm::mr::device_memory_resource* mr,
                         rmm::mr::device_memory_resource* temp_mr)
    : contiguous_split_state(input, splits, 0, stream, mr, temp_mr)
  {
  }

  bool has_next() const { return !is_empty && chunk_iter_state->has_more_copies(); }

  std::size_t get_total_contiguous_size() const
  {
    return is_empty ? 0 : chunk_iter_state->total_size;
  }

  std::vector<packed_table> contiguous_split()
  {
    CUDF_EXPECTS(user_buffer_size == 0, "Cannot contiguous split with a user buffer");
    if (is_empty || input.num_columns() == 0) { return make_packed_tables(); }

    auto const num_batches_total =
      std::get<1>(chunk_iter_state->get_current_starting_index_and_buff_count());

    // perform the copy.
    copy_data(num_batches_total,
              0 /* starting at buffer for single-shot 0*/,
              src_and_dst_pointers->d_src_bufs,
              src_and_dst_pointers->d_dst_bufs,
              chunk_iter_state->d_batched_dst_buf_info,
              nullptr,
              stream);

    // these "orig" dst_buf_info pointers describe the prior-to-batching destination
    // buffers per partition
    auto d_orig_dst_buf_info = partition_buf_size_and_dst_buf_info->d_dst_buf_info;
    auto h_orig_dst_buf_info = partition_buf_size_and_dst_buf_info->h_dst_buf_info;

    // postprocess valid_counts: apply the valid counts computed by copy_data for each
    // batch back to the original dst_buf_infos
    auto const keys = cudf::detail::make_counting_transform_iterator(
      0, out_to_in_index_function{chunk_iter_state->d_batch_offsets.begin(), (int)num_bufs});

    auto values = thrust::make_transform_iterator(
      chunk_iter_state->d_batched_dst_buf_info.begin(),
      cuda::proclaim_return_type<size_type>(
        [] __device__(dst_buf_info const& info) { return info.valid_count; }));

    thrust::reduce_by_key(rmm::exec_policy(stream, temp_mr),
                          keys,
                          keys + num_batches_total,
                          values,
                          thrust::make_discard_iterator(),
                          dst_valid_count_output_iterator{d_orig_dst_buf_info});

    CUDF_CUDA_TRY(hipMemcpyAsync(h_orig_dst_buf_info,
                                  d_orig_dst_buf_info,
                                  partition_buf_size_and_dst_buf_info->dst_buf_info_size,
                                  hipMemcpyDefault,
                                  stream.value()));

    stream.synchronize();

    // not necessary for the non-chunked case, but it makes it so further calls to has_next
    // return false, just in case
    chunk_iter_state->advance_iteration();

    return make_packed_tables();
  }

  cudf::size_type contiguous_split_chunk(cudf::device_span<uint8_t> const& user_buffer)
  {
    CUDF_FUNC_RANGE();
    CUDF_EXPECTS(
      user_buffer.size() == user_buffer_size,
      "Cannot use a device span smaller than the output buffer size configured at instantiation!");
    CUDF_EXPECTS(has_next(), "Cannot call contiguous_split_chunk with has_next() == false!");

    auto [starting_batch, num_batches_to_copy] =
      chunk_iter_state->get_current_starting_index_and_buff_count();

    // perform the copy.
    copy_data(num_batches_to_copy,
              starting_batch,
              src_and_dst_pointers->d_src_bufs,
              src_and_dst_pointers->d_dst_bufs,
              chunk_iter_state->d_batched_dst_buf_info,
              user_buffer.data(),
              stream);

    // We do not need to post-process null counts since the null count info is
    // taken from the source table in the contiguous_split_chunk case (no splits)
    return chunk_iter_state->advance_iteration();
  }

  std::unique_ptr<std::vector<uint8_t>> build_packed_column_metadata()
  {
    CUDF_EXPECTS(num_partitions == 1, "build_packed_column_metadata supported only without splits");

    if (input.num_columns() == 0) { return std::unique_ptr<std::vector<uint8_t>>(); }

    if (is_empty) {
      // this is a bit ugly, but it was done to re-use make_empty_packed_table between the
      // regular contiguous_split and chunked_pack cases.
      auto empty_packed_tables = std::move(make_empty_packed_table().front());
      return std::move(empty_packed_tables.data.metadata);
    }

    auto& h_dst_buf_info  = partition_buf_size_and_dst_buf_info->h_dst_buf_info;
    auto cur_dst_buf_info = h_dst_buf_info;
    detail::metadata_builder mb{input.num_columns()};

    populate_metadata(input.begin(), input.end(), cur_dst_buf_info, mb);

    return std::make_unique<std::vector<uint8_t>>(std::move(mb.build()));
  }

 private:
  contiguous_split_state(cudf::table_view const& input,
                         std::vector<size_type> const& splits,
                         std::size_t user_buffer_size,
                         rmm::cuda_stream_view stream,
                         rmm::mr::device_memory_resource* mr,
                         rmm::mr::device_memory_resource* temp_mr)
    : input(input),
      user_buffer_size(user_buffer_size),
      stream(stream),
      mr(mr),
      temp_mr(temp_mr),
      is_empty{check_inputs(input, splits)},
      num_partitions{splits.size() + 1},
      num_src_bufs{count_src_bufs(input.begin(), input.end())},
      num_bufs{num_src_bufs * num_partitions}
  {
    // if the table we are about to contig split is empty, we have special
    // handling where metadata is produced and a 0-byte contiguous buffer
    // is the result.
    if (is_empty) { return; }

    // First pass over the source tables to generate a `dst_buf_info` per split and column buffer
    // (`num_bufs`). After this, contiguous_split uses `dst_buf_info` to further subdivide the work
    // into 1MB batches in `compute_batches`
    partition_buf_size_and_dst_buf_info = std::move(
      compute_splits(input, splits, num_partitions, num_src_bufs, num_bufs, stream, temp_mr));

    // Second pass: uses `dst_buf_info` to break down the work into 1MB batches.
    chunk_iter_state = compute_batches(num_bufs,
                                       partition_buf_size_and_dst_buf_info->d_dst_buf_info,
                                       partition_buf_size_and_dst_buf_info->h_buf_sizes,
                                       num_partitions,
                                       user_buffer_size,
                                       stream,
                                       temp_mr);

    // allocate output partition buffers, in the non-chunked case
    if (user_buffer_size == 0) {
      out_buffers.reserve(num_partitions);
      auto h_buf_sizes = partition_buf_size_and_dst_buf_info->h_buf_sizes;
      std::transform(h_buf_sizes,
                     h_buf_sizes + num_partitions,
                     std::back_inserter(out_buffers),
                     [stream = stream, mr = mr](std::size_t bytes) {
                       return rmm::device_buffer{bytes, stream, mr};
                     });
    }

    src_and_dst_pointers = std::move(setup_src_and_dst_pointers(
      input, num_partitions, num_src_bufs, out_buffers, stream, temp_mr));
  }

  std::vector<packed_table> make_packed_tables()
  {
    if (input.num_columns() == 0) { return std::vector<packed_table>(); }
    if (is_empty) { return make_empty_packed_table(); }
    std::vector<packed_table> result;
    result.reserve(num_partitions);
    std::vector<column_view> cols;
    cols.reserve(input.num_columns());

    auto& h_dst_buf_info = partition_buf_size_and_dst_buf_info->h_dst_buf_info;
    auto& h_dst_bufs     = src_and_dst_pointers->h_dst_bufs;

    auto cur_dst_buf_info = h_dst_buf_info;
    detail::metadata_builder mb(input.num_columns());

    for (std::size_t idx = 0; idx < num_partitions; idx++) {
      // traverse the buffers and build the columns.
      cur_dst_buf_info = build_output_columns(input.begin(),
                                              input.end(),
                                              cur_dst_buf_info,
                                              std::back_inserter(cols),
                                              h_dst_bufs[idx],
                                              mb);

      // pack the columns
      result.emplace_back(packed_table{
        cudf::table_view{cols},
        packed_columns{std::make_unique<std::vector<uint8_t>>(mb.build()),
                       std::make_unique<rmm::device_buffer>(std::move(out_buffers[idx]))}});

      cols.clear();
      mb.clear();
    }

    return result;
  }

  std::vector<packed_table> make_empty_packed_table()
  {
    // sanitize the inputs (to handle corner cases like sliced tables)
    std::vector<cudf::column_view> empty_column_views;
    empty_column_views.reserve(input.num_columns());
    std::transform(input.begin(),
                   input.end(),
                   std::back_inserter(empty_column_views),
                   [](column_view const& col) { return cudf::empty_like(col)->view(); });

    table_view empty_inputs(empty_column_views);

    // build the empty results
    std::vector<packed_table> result;
    result.reserve(num_partitions);
    auto const iter = thrust::make_counting_iterator(0);
    std::transform(iter,
                   iter + num_partitions,
                   std::back_inserter(result),
                   [&empty_inputs](int partition_index) {
                     return packed_table{empty_inputs,
                                         packed_columns{std::make_unique<std::vector<uint8_t>>(
                                                          pack_metadata(empty_inputs, nullptr, 0)),
                                                        std::make_unique<rmm::device_buffer>()}};
                   });

    return result;
  }

  cudf::table_view const input;        ///< The input table_view to operate on
  std::size_t const user_buffer_size;  ///< The size of the user buffer for the chunked_pack case
  rmm::cuda_stream_view const stream;
  rmm::mr::device_memory_resource* const mr;  ///< The memory resource for any data returned

  // this resource defaults to `mr` for the contiguous_split case, but it can be useful for the
  // `chunked_pack` case to allocate scratch/temp memory in a pool
  rmm::mr::device_memory_resource* const temp_mr;  ///< The memory resource for scratch/temp space

  // whether the table was empty to begin with (0 rows or 0 columns) and should be metadata-only
  bool const is_empty;  ///< True if the source table has 0 rows or 0 columns

  // This can be 1 if `contiguous_split` is just packing and not splitting
  std::size_t const num_partitions;  ///< The number of partitions to produce

  size_type const num_src_bufs;  ///< Number of source buffers including children

  std::size_t const num_bufs;  ///< Number of source buffers including children * number of splits

  std::unique_ptr<packed_partition_buf_size_and_dst_buf_info>
    partition_buf_size_and_dst_buf_info;  ///< Per-partition buffer size and destination buffer info

  std::unique_ptr<packed_src_and_dst_pointers>
    src_and_dst_pointers;  ///< Src. and dst. pointers for `copy_partition`

  //
  // State around the chunked pattern
  //

  // chunked_pack will have 1 or more "chunks" to iterate on, defined in chunk_iter_state
  // contiguous_split will have a single "chunk" in chunk_iter_state, so no iteration.
  std::unique_ptr<chunk_iteration_state>
    chunk_iter_state;  ///< State object for chunk iteration state

  // Two API usages are allowed:
  //  - `chunked_pack`: for this mode, the user will provide a buffer that must be at least 1MB.
  //    The behavior is "chunked" in that it will contiguously copy up until the user specified
  //    `user_buffer_size` limit, exposing a next() call for the user to invoke. Note that in this
  //    mode, no partitioning occurs, hence the name "pack".
  //
  //  - `contiguous_split` (default): when the user doesn't provide their own buffer,
  //    `contiguous_split` will allocate a buffer per partition and will place contiguous results in
  //    each buffer.
  //
  std::vector<rmm::device_buffer>
    out_buffers;  ///< Buffers allocated for a regular `contiguous_split`
};

std::vector<packed_table> contiguous_split(cudf::table_view const& input,
                                           std::vector<size_type> const& splits,
                                           rmm::cuda_stream_view stream,
                                           rmm::mr::device_memory_resource* mr)
{
  // `temp_mr` is the same as `mr` for contiguous_split as it allocates all
  // of its memory from the default memory resource in cuDF
  auto temp_mr = mr;
  auto state   = contiguous_split_state(input, splits, stream, mr, temp_mr);
  return state.contiguous_split();
}

};  // namespace detail

std::vector<packed_table> contiguous_split(cudf::table_view const& input,
                                           std::vector<size_type> const& splits,
                                           rmm::mr::device_memory_resource* mr)
{
  CUDF_FUNC_RANGE();
  return detail::contiguous_split(input, splits, cudf::get_default_stream(), mr);
}

chunked_pack::chunked_pack(cudf::table_view const& input,
                           std::size_t user_buffer_size,
                           rmm::mr::device_memory_resource* temp_mr)
{
  CUDF_EXPECTS(user_buffer_size >= desired_batch_size,
               "The output buffer size must be at least 1MB in size");
  // We pass `nullptr` for the first `mr` in `contiguous_split_state` to indicate
  // that it does not allocate any user-bound data for the `chunked_pack` case.
  state = std::make_unique<detail::contiguous_split_state>(
    input, user_buffer_size, cudf::get_default_stream(), nullptr, temp_mr);
}

// required for the unique_ptr to work with a incomplete type (contiguous_split_state)
chunked_pack::~chunked_pack() = default;

std::size_t chunked_pack::get_total_contiguous_size() const
{
  return state->get_total_contiguous_size();
}

bool chunked_pack::has_next() const { return state->has_next(); }

std::size_t chunked_pack::next(cudf::device_span<uint8_t> const& user_buffer)
{
  return state->contiguous_split_chunk(user_buffer);
}

std::unique_ptr<std::vector<uint8_t>> chunked_pack::build_metadata() const
{
  return state->build_packed_column_metadata();
}

std::unique_ptr<chunked_pack> chunked_pack::create(cudf::table_view const& input,
                                                   std::size_t user_buffer_size,
                                                   rmm::mr::device_memory_resource* temp_mr)
{
  return std::make_unique<chunked_pack>(input, user_buffer_size, temp_mr);
}

};  // namespace cudf
