/*
 * Copyright (c) 2019-2020, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <cudf/copying.hpp>
#include <cudf/detail/copy.hpp>
#include <cudf/detail/copy_if_else.cuh>
#include <cudf/detail/iterator.cuh>
#include <cudf/detail/nvtx/ranges.hpp>
#include <cudf/strings/string_view.cuh>
#include "cudf/fixed_point/fixed_point.hpp"

namespace cudf {
namespace detail {
namespace {
/**
 * @brief Specialization of copy_if_else_functor for string_views.
 */
template <typename T, typename Left, typename Right, typename Filter>
struct copy_if_else_functor_impl {
  std::unique_ptr<column> operator()(Left const& lhs,
                                     Right const& rhs,
                                     size_type size,
                                     bool left_nullable,
                                     bool right_nullable,
                                     Filter filter,
                                     rmm::mr::device_memory_resource* mr,
                                     hipStream_t stream)
  {
    if (left_nullable) {
      if (right_nullable) {
        auto lhs_iter = cudf::detail::make_pair_iterator<T, true>(lhs);
        auto rhs_iter = cudf::detail::make_pair_iterator<T, true>(rhs);
        return detail::copy_if_else(true, lhs_iter, lhs_iter + size, rhs_iter, filter, mr, stream);
      }
      auto lhs_iter = cudf::detail::make_pair_iterator<T, true>(lhs);
      auto rhs_iter = cudf::detail::make_pair_iterator<T, false>(rhs);
      return detail::copy_if_else(true, lhs_iter, lhs_iter + size, rhs_iter, filter, mr, stream);
    }
    if (right_nullable) {
      auto lhs_iter = cudf::detail::make_pair_iterator<T, false>(lhs);
      auto rhs_iter = cudf::detail::make_pair_iterator<T, true>(rhs);
      return detail::copy_if_else(true, lhs_iter, lhs_iter + size, rhs_iter, filter, mr, stream);
    }
    auto lhs_iter = cudf::detail::make_pair_iterator<T, false>(lhs);
    auto rhs_iter = cudf::detail::make_pair_iterator<T, false>(rhs);
    return detail::copy_if_else(false, lhs_iter, lhs_iter + size, rhs_iter, filter, mr, stream);
  }
};

/**
 * @brief Specialization of copy_if_else_functor for string_views.
 */
template <typename Left, typename Right, typename Filter>
struct copy_if_else_functor_impl<string_view, Left, Right, Filter> {
  std::unique_ptr<column> operator()(Left const& lhs,
                                     Right const& rhs,
                                     size_type size,
                                     bool left_nullable,
                                     bool right_nullable,
                                     Filter filter,
                                     rmm::mr::device_memory_resource* mr,
                                     hipStream_t stream)
  {
    using T = string_view;

    if (left_nullable) {
      if (right_nullable) {
        auto lhs_iter = cudf::detail::make_pair_iterator<T, true>(lhs);
        auto rhs_iter = cudf::detail::make_pair_iterator<T, true>(rhs);
        return strings::detail::copy_if_else(
          lhs_iter, lhs_iter + size, rhs_iter, filter, mr, stream);
      }
      auto lhs_iter = cudf::detail::make_pair_iterator<T, true>(lhs);
      auto rhs_iter = cudf::detail::make_pair_iterator<T, false>(rhs);
      return strings::detail::copy_if_else(lhs_iter, lhs_iter + size, rhs_iter, filter, mr, stream);
    }
    if (right_nullable) {
      auto lhs_iter = cudf::detail::make_pair_iterator<T, false>(lhs);
      auto rhs_iter = cudf::detail::make_pair_iterator<T, true>(rhs);
      return strings::detail::copy_if_else(lhs_iter, lhs_iter + size, rhs_iter, filter, mr, stream);
    }
    auto lhs_iter = cudf::detail::make_pair_iterator<T, false>(lhs);
    auto rhs_iter = cudf::detail::make_pair_iterator<T, false>(rhs);
    return strings::detail::copy_if_else(lhs_iter, lhs_iter + size, rhs_iter, filter, mr, stream);
  }
};

/**
 * @brief Specialization of copy_if_else_functor for list_views.
 */
template <typename Left, typename Right, typename Filter>
struct copy_if_else_functor_impl<list_view, Left, Right, Filter> {
  std::unique_ptr<column> operator()(Left const& lhs,
                                     Right const& rhs,
                                     size_type size,
                                     bool left_nullable,
                                     bool right_nullable,
                                     Filter filter,
                                     rmm::mr::device_memory_resource* mr,
                                     hipStream_t stream)
  {
    CUDF_FAIL("copy_if_else not supported for list_view yet");
  }
};

template <typename Left, typename Right, typename Filter>
struct copy_if_else_functor_impl<struct_view, Left, Right, Filter> {
  std::unique_ptr<column> operator()(Left const& lhs,
                                     Right const& rhs,
                                     size_type size,
                                     bool left_nullable,
                                     bool right_nullable,
                                     Filter filter,
                                     rmm::mr::device_memory_resource* mr,
                                     hipStream_t stream)
  {
    CUDF_FAIL("copy_if_else not supported for struct_view yet");
  }
};

/**
 * @brief Specialization of copy_if_else_functor for decimal32.
 */
template <typename Left, typename Right, typename Filter>
struct copy_if_else_functor_impl<numeric::decimal32, Left, Right, Filter> {
  std::unique_ptr<column> operator()(Left const& lhs,
                                     Right const& rhs,
                                     size_type size,
                                     bool left_nullable,
                                     bool right_nullable,
                                     Filter filter,
                                     rmm::mr::device_memory_resource* mr,
                                     hipStream_t stream)
  {
    CUDF_FAIL("copy_if_else not supported for decimal32 yet");
  }
};

/**
 * @brief Specialization of copy_if_else_functor for decimal64.
 */
template <typename Left, typename Right, typename Filter>
struct copy_if_else_functor_impl<numeric::decimal64, Left, Right, Filter> {
  std::unique_ptr<column> operator()(Left const& lhs,
                                     Right const& rhs,
                                     size_type size,
                                     bool left_nullable,
                                     bool right_nullable,
                                     Filter filter,
                                     rmm::mr::device_memory_resource* mr,
                                     hipStream_t stream)
  {
    CUDF_FAIL("copy_if_else not supported for decimal64 yet");
  }
};

/**
 * @brief Functor called by the `type_dispatcher` to invoke copy_if_else on combinations
 *        of column_view and scalar
 */
struct copy_if_else_functor {
  template <typename T, typename Left, typename Right, typename Filter>
  std::unique_ptr<column> operator()(Left const& lhs,
                                     Right const& rhs,
                                     size_type size,
                                     bool left_nullable,
                                     bool right_nullable,
                                     Filter filter,
                                     rmm::mr::device_memory_resource* mr,
                                     hipStream_t stream)
  {
    copy_if_else_functor_impl<T, Left, Right, Filter> copier{};
    return copier(lhs, rhs, size, left_nullable, right_nullable, filter, mr, stream);
  }
};

// wrap up boolean_mask into a filter lambda
template <typename Left, typename Right>
std::unique_ptr<column> copy_if_else(Left const& lhs,
                                     Right const& rhs,
                                     bool left_nullable,
                                     bool right_nullable,
                                     column_view const& boolean_mask,
                                     rmm::mr::device_memory_resource* mr,
                                     hipStream_t stream)
{
  CUDF_EXPECTS(lhs.type() == rhs.type(), "Both inputs must be of the same type");
  CUDF_EXPECTS(boolean_mask.type() == data_type(type_id::BOOL8),
               "Boolean mask column must be of type type_id::BOOL8");

  if (boolean_mask.is_empty()) { return cudf::make_empty_column(lhs.type()); }

  auto bool_mask_device_p             = column_device_view::create(boolean_mask);
  column_device_view bool_mask_device = *bool_mask_device_p;

  if (boolean_mask.has_nulls()) {
    auto filter = [bool_mask_device] __device__(cudf::size_type i) {
      return bool_mask_device.is_valid_nocheck(i) and bool_mask_device.element<bool>(i);
    };
    return cudf::type_dispatcher(lhs.type(),
                                 copy_if_else_functor{},
                                 lhs,
                                 rhs,
                                 boolean_mask.size(),
                                 left_nullable,
                                 right_nullable,
                                 filter,
                                 mr,
                                 stream);
  } else {
    auto filter = [bool_mask_device] __device__(cudf::size_type i) {
      return bool_mask_device.element<bool>(i);
    };
    return cudf::type_dispatcher(lhs.type(),
                                 copy_if_else_functor{},
                                 lhs,
                                 rhs,
                                 boolean_mask.size(),
                                 left_nullable,
                                 right_nullable,
                                 filter,
                                 mr,
                                 stream);
  }
}

};  // namespace

std::unique_ptr<column> copy_if_else(column_view const& lhs,
                                     column_view const& rhs,
                                     column_view const& boolean_mask,
                                     rmm::mr::device_memory_resource* mr,
                                     hipStream_t stream)
{
  CUDF_EXPECTS(boolean_mask.size() == lhs.size(),
               "Boolean mask column must be the same size as lhs and rhs columns");
  CUDF_EXPECTS(lhs.size() == rhs.size(), "Both columns must be of the size");
  return copy_if_else(*column_device_view::create(lhs),
                      *column_device_view::create(rhs),
                      lhs.has_nulls(),
                      rhs.has_nulls(),
                      boolean_mask,
                      mr,
                      stream);
}

std::unique_ptr<column> copy_if_else(scalar const& lhs,
                                     column_view const& rhs,
                                     column_view const& boolean_mask,
                                     rmm::mr::device_memory_resource* mr,
                                     hipStream_t stream)
{
  CUDF_EXPECTS(boolean_mask.size() == rhs.size(),
               "Boolean mask column must be the same size as rhs column");
  return copy_if_else(lhs,
                      *column_device_view::create(rhs),
                      !lhs.is_valid(),
                      rhs.has_nulls(),
                      boolean_mask,
                      mr,
                      stream);
}

std::unique_ptr<column> copy_if_else(column_view const& lhs,
                                     scalar const& rhs,
                                     column_view const& boolean_mask,
                                     rmm::mr::device_memory_resource* mr,
                                     hipStream_t stream)
{
  CUDF_EXPECTS(boolean_mask.size() == lhs.size(),
               "Boolean mask column must be the same size as lhs column");
  return copy_if_else(*column_device_view::create(lhs),
                      rhs,
                      lhs.has_nulls(),
                      !rhs.is_valid(),
                      boolean_mask,
                      mr,
                      stream);
}

std::unique_ptr<column> copy_if_else(scalar const& lhs,
                                     scalar const& rhs,
                                     column_view const& boolean_mask,
                                     rmm::mr::device_memory_resource* mr,
                                     hipStream_t stream)
{
  return copy_if_else(lhs, rhs, !lhs.is_valid(), !rhs.is_valid(), boolean_mask, mr, stream);
}

};  // namespace detail

std::unique_ptr<column> copy_if_else(column_view const& lhs,
                                     column_view const& rhs,
                                     column_view const& boolean_mask,
                                     rmm::mr::device_memory_resource* mr)
{
  CUDF_FUNC_RANGE();
  return detail::copy_if_else(lhs, rhs, boolean_mask, mr);
}

std::unique_ptr<column> copy_if_else(scalar const& lhs,
                                     column_view const& rhs,
                                     column_view const& boolean_mask,
                                     rmm::mr::device_memory_resource* mr)
{
  CUDF_FUNC_RANGE();
  return detail::copy_if_else(lhs, rhs, boolean_mask, mr);
}

std::unique_ptr<column> copy_if_else(column_view const& lhs,
                                     scalar const& rhs,
                                     column_view const& boolean_mask,
                                     rmm::mr::device_memory_resource* mr)
{
  CUDF_FUNC_RANGE();
  return detail::copy_if_else(lhs, rhs, boolean_mask, mr);
}

std::unique_ptr<column> copy_if_else(scalar const& lhs,
                                     scalar const& rhs,
                                     column_view const& boolean_mask,
                                     rmm::mr::device_memory_resource* mr)
{
  CUDF_FUNC_RANGE();
  return detail::copy_if_else(lhs, rhs, boolean_mask, mr);
}

}  // namespace cudf
