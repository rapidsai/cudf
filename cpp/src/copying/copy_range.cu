/*
 * Copyright (c) 2019-2020, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <cudf/column/column.hpp>
#include <cudf/column/column_device_view.cuh>
#include <cudf/column/column_factories.hpp>
#include <cudf/column/column_view.hpp>
#include <cudf/copying.hpp>
#include <cudf/detail/copy_range.cuh>
#include <cudf/detail/iterator.cuh>
#include <cudf/detail/nvtx/ranges.hpp>
#include <cudf/dictionary/detail/update_keys.hpp>
#include <cudf/dictionary/dictionary_column_view.hpp>
#include <cudf/dictionary/dictionary_factories.hpp>
#include <cudf/strings/detail/copy_range.cuh>
#include <cudf/strings/strings_column_view.hpp>
#include <cudf/types.hpp>
#include <cudf/utilities/error.hpp>
#include <cudf/utilities/traits.hpp>

#include <thrust/iterator/constant_iterator.h>

#include <hip/hip_runtime.h>

#include <memory>

namespace {
template <typename T>
void in_place_copy_range(cudf::column_view const& source,
                         cudf::mutable_column_view& target,
                         cudf::size_type source_begin,
                         cudf::size_type source_end,
                         cudf::size_type target_begin,
                         hipStream_t stream = 0)
{
  auto p_source_device_view = cudf::column_device_view::create(source, stream);
  if (source.has_nulls()) {
    cudf::detail::copy_range(
      cudf::detail::make_null_replacement_iterator<T>(*p_source_device_view, T()) + source_begin,
      cudf::detail::make_validity_iterator(*p_source_device_view) + source_begin,
      target,
      target_begin,
      target_begin + (source_end - source_begin),
      stream);
  } else {
    cudf::detail::copy_range(p_source_device_view->begin<T>() + source_begin,
                             thrust::make_constant_iterator(true),  // dummy
                             target,
                             target_begin,
                             target_begin + (source_end - source_begin),
                             stream);
  }
}

struct in_place_copy_range_dispatch {
  cudf::column_view const& source;
  cudf::mutable_column_view& target;

  template <typename T>
  std::enable_if_t<cudf::is_fixed_width<T>(), void> operator()(cudf::size_type source_begin,
                                                               cudf::size_type source_end,
                                                               cudf::size_type target_begin,
                                                               hipStream_t stream = 0)
  {
    in_place_copy_range<T>(source, target, source_begin, source_end, target_begin, stream);
  }

  template <typename T>
  std::enable_if_t<not cudf::is_fixed_width<T>(), void> operator()(cudf::size_type source_begin,
                                                                   cudf::size_type source_end,
                                                                   cudf::size_type target_begin,
                                                                   hipStream_t stream = 0)
  {
    CUDF_FAIL("in-place copy does not work for variable width types.");
  }
};

struct out_of_place_copy_range_dispatch {
  cudf::column_view const& source;
  cudf::column_view const& target;

  template <typename T>
  std::unique_ptr<cudf::column> operator()(
    cudf::size_type source_begin,
    cudf::size_type source_end,
    cudf::size_type target_begin,
    rmm::mr::device_memory_resource* mr = rmm::mr::get_current_device_resource(),
    hipStream_t stream                 = 0)
  {
    auto p_ret = std::make_unique<cudf::column>(target, stream, mr);
    if ((!p_ret->nullable()) && source.has_nulls(source_begin, source_end)) {
      p_ret->set_null_mask(
        cudf::create_null_mask(p_ret->size(), cudf::mask_state::ALL_VALID, stream, mr), 0);
    }

    if (source_end != source_begin) {  // otherwise no-op
      auto ret_view = p_ret->mutable_view();
      in_place_copy_range<T>(source, ret_view, source_begin, source_end, target_begin, stream);
    }

    return p_ret;
  }
};

template <>
std::unique_ptr<cudf::column> out_of_place_copy_range_dispatch::operator()<cudf::string_view>(
  cudf::size_type source_begin,
  cudf::size_type source_end,
  cudf::size_type target_begin,
  rmm::mr::device_memory_resource* mr,
  hipStream_t stream)
{
  auto target_end           = target_begin + (source_end - source_begin);
  auto p_source_device_view = cudf::column_device_view::create(source, stream);
  if (source.has_nulls()) {
    return cudf::strings::detail::copy_range(
      cudf::detail::make_null_replacement_iterator<cudf::string_view>(*p_source_device_view,
                                                                      cudf::string_view()) +
        source_begin,
      cudf::detail::make_validity_iterator(*p_source_device_view) + source_begin,
      cudf::strings_column_view(target),
      target_begin,
      target_end,
      mr,
      stream);
  } else {
    return cudf::strings::detail::copy_range(
      p_source_device_view->begin<cudf::string_view>() + source_begin,
      thrust::make_constant_iterator(true),
      cudf::strings_column_view(target),
      target_begin,
      target_end,
      mr,
      stream);
  }
}

template <>
std::unique_ptr<cudf::column> out_of_place_copy_range_dispatch::operator()<numeric::decimal64>(
  cudf::size_type source_begin,
  cudf::size_type source_end,
  cudf::size_type target_begin,
  rmm::mr::device_memory_resource* mr,
  hipStream_t stream)
{
  CUDF_FAIL("decimal64 type not supported");
}

template <>
std::unique_ptr<cudf::column> out_of_place_copy_range_dispatch::operator()<numeric::decimal32>(
  cudf::size_type source_begin,
  cudf::size_type source_end,
  cudf::size_type target_begin,
  rmm::mr::device_memory_resource* mr,
  hipStream_t stream)
{
  CUDF_FAIL("decimal32 type not supported");
}

template <>
std::unique_ptr<cudf::column> out_of_place_copy_range_dispatch::operator()<cudf::dictionary32>(
  cudf::size_type source_begin,
  cudf::size_type source_end,
  cudf::size_type target_begin,
  rmm::mr::device_memory_resource* mr,
  hipStream_t stream)
{
  // check the keys in the source and target
  cudf::dictionary_column_view const dict_source(source);
  cudf::dictionary_column_view const dict_target(target);
  CUDF_EXPECTS(dict_source.keys().type() == dict_target.keys().type(),
               "dictionary keys must be the same type");

  // combine keys so both dictionaries have the same set
  auto target_matched =
    cudf::dictionary::detail::add_keys(dict_target, dict_source.keys(), mr, stream);
  auto const target_view = cudf::dictionary_column_view(target_matched->view());
  auto source_matched    = cudf::dictionary::detail::set_keys(
    dict_source, target_view.keys(), rmm::mr::get_current_device_resource(), stream);
  auto const source_view = cudf::dictionary_column_view(source_matched->view());

  // build the new indices by calling in_place_copy_range on just the indices
  auto const source_indices = source_view.get_indices_annotated();
  auto target_contents      = target_matched->release();
  auto target_indices(std::move(target_contents.children.front()));
  cudf::mutable_column_view new_indices(
    target_indices->type(),
    dict_target.size(),
    target_indices->mutable_view().head(),
    static_cast<cudf::bitmask_type*>(target_contents.null_mask->data()),
    dict_target.null_count());
  cudf::type_dispatcher(new_indices.type(),
                        in_place_copy_range_dispatch{source_indices, new_indices},
                        source_begin,
                        source_end,
                        target_begin,
                        stream);
  auto null_count = new_indices.null_count();
  auto indices_column =
    std::make_unique<cudf::column>(new_indices.type(),
                                   new_indices.size(),
                                   std::move(*(target_indices->release().data.release())),
                                   rmm::device_buffer{0, stream, mr},
                                   0);

  // take the keys from the matched column allocated using mr
  auto keys_column(std::move(target_contents.children.back()));

  // create column with keys_column and indices_column
  return make_dictionary_column(std::move(keys_column),
                                std::move(indices_column),
                                std::move(*(target_contents.null_mask.release())),
                                null_count);
}

template <>
std::unique_ptr<cudf::column> out_of_place_copy_range_dispatch::operator()<cudf::list_view>(
  cudf::size_type source_begin,
  cudf::size_type source_end,
  cudf::size_type target_begin,
  rmm::mr::device_memory_resource* mr,
  hipStream_t stream)
{
  CUDF_FAIL("list_view type not supported");
}

}  // namespace

namespace cudf {
namespace detail {
void copy_range_in_place(column_view const& source,
                         mutable_column_view& target,
                         size_type source_begin,
                         size_type source_end,
                         size_type target_begin,
                         hipStream_t stream)
{
  CUDF_EXPECTS(cudf::is_fixed_width(target.type()) == true,
               "In-place copy_range does not support variable-sized types.");
  CUDF_EXPECTS((source_begin >= 0) && (source_end <= source.size()) &&
                 (source_begin <= source_end) && (target_begin >= 0) &&
                 (target_begin <= target.size() - (source_end - source_begin)),
               "Range is out of bounds.");
  CUDF_EXPECTS(target.type() == source.type(), "Data type mismatch.");
  CUDF_EXPECTS((target.nullable() == true) || (source.has_nulls() == false),
               "target should be nullable if source has null values.");

  if (source_end != source_begin) {  // otherwise no-op
    cudf::type_dispatcher(target.type(),
                          in_place_copy_range_dispatch{source, target},
                          source_begin,
                          source_end,
                          target_begin,
                          stream);
  }
}

std::unique_ptr<column> copy_range(column_view const& source,
                                   column_view const& target,
                                   size_type source_begin,
                                   size_type source_end,
                                   size_type target_begin,
                                   rmm::mr::device_memory_resource* mr,
                                   hipStream_t stream)
{
  CUDF_EXPECTS((source_begin >= 0) && (source_end <= source.size()) &&
                 (source_begin <= source_end) && (target_begin >= 0) &&
                 (target_begin <= target.size() - (source_end - source_begin)),
               "Range is out of bounds.");
  CUDF_EXPECTS(target.type() == source.type(), "Data type mismatch.");

  return cudf::type_dispatcher(target.type(),
                               out_of_place_copy_range_dispatch{source, target},
                               source_begin,
                               source_end,
                               target_begin,
                               mr,
                               stream);
}

}  // namespace detail

void copy_range_in_place(column_view const& source,
                         mutable_column_view& target,
                         size_type source_begin,
                         size_type source_end,
                         size_type target_begin)
{
  CUDF_FUNC_RANGE();
  return detail::copy_range_in_place(source, target, source_begin, source_end, target_begin, 0);
}

std::unique_ptr<column> copy_range(column_view const& source,
                                   column_view const& target,
                                   size_type source_begin,
                                   size_type source_end,
                                   size_type target_begin,
                                   rmm::mr::device_memory_resource* mr)
{
  CUDF_FUNC_RANGE();
  return detail::copy_range(source, target, source_begin, source_end, target_begin, mr, 0);
}

}  // namespace cudf
