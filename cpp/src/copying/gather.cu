#include "hip/hip_runtime.h"
#include <cudf/detail/gather.cuh>
#include <cudf/detail/gather.hpp>
#include <cudf/types.hpp>
#include <cudf/column/column_view.hpp>
#include <cudf/utilities/type_dispatcher.hpp>
#include <cudf/table/table_view.hpp>
#include <cudf/table/table.hpp>
#include <cudf/copying.hpp>
#include <utilities/legacy/error_utils.hpp>

#include <rmm/thrust_rmm_allocator.h>
#include <thrust/count.h>

#include <memory>


namespace cudf {
namespace experimental {
namespace detail {


struct dispatch_map_type {
  template <typename map_type, std::enable_if_t<std::is_integral<map_type>::value
    and not std::is_same<map_type, cudf::experimental::bool8>::value>* = nullptr>
  std::unique_ptr<table> operator()(table_view const& source_table,
				    column_view const& gather_map,
				    size_type num_destination_rows, bool check_bounds,
				    bool ignore_out_of_bounds,
				    bool allow_negative_indices = false,
				    rmm::mr::device_memory_resource* mr = rmm::mr::get_default_resource(),
				    hipStream_t stream = 0)
  {
    std::unique_ptr<table> destination_table;

    if (check_bounds) {
      cudf::size_type begin = (allow_negative_indices) ? -source_table.num_rows() : 0;
      CUDF_EXPECTS(
	  num_destination_rows == thrust::count_if(
	      rmm::exec_policy()->on(0),
	      gather_map.begin<map_type>(),
	      gather_map.end<map_type>(),
	      bounds_checker<map_type>{begin, source_table.num_rows()}),
	  "Index out of bounds.");
    }

    if (allow_negative_indices) {
      destination_table =
	gather(source_table,
	       thrust::make_transform_iterator(
					       gather_map.begin<map_type>(),
					       index_converter<map_type>{source_table.num_rows()}),
	       thrust::make_transform_iterator(
					       gather_map.end<map_type>(),
					       index_converter<map_type>{source_table.num_rows()}),
	       ignore_out_of_bounds,
	       mr,
	       stream
	     );
    }
    else {
      destination_table =
	gather(source_table,
	       gather_map.begin<map_type>(),
	       gather_map.end<map_type>(),
	       ignore_out_of_bounds,
	       mr,
	       stream
	       );
    }

    return destination_table;
  }

  template <typename map_type, std::enable_if_t<not std::is_integral<map_type>::value
    or std::is_same<map_type, cudf::experimental::bool8>::value>* = nullptr>
  std::unique_ptr<table> operator()(table_view const& source_table, column_view const& gather_map,
				    size_type num_destination_rows, bool check_bounds,
				    bool ignore_out_of_bounds, bool allow_negative_indices = false,
				    rmm::mr::device_memory_resource* mr = rmm::mr::get_default_resource(),
				    hipStream_t stream = 0) {
    CUDF_FAIL("Gather map must be an integral type.");
  }
};

std::unique_ptr<table> gather(table_view const& source_table, column_view const& gather_map,
			      bool check_bounds, bool ignore_out_of_bounds,
			      bool allow_negative_indices,
			      rmm::mr::device_memory_resource* mr,
			      hipStream_t stream) {

  CUDF_EXPECTS(gather_map.has_nulls() == false, "gather_map contains nulls");

  std::unique_ptr<table> destination_table =
    cudf::experimental::type_dispatcher(gather_map.type(), dispatch_map_type{},
					source_table, gather_map,
					gather_map.size(),
					check_bounds, ignore_out_of_bounds,
					allow_negative_indices,
					mr,
					stream);

  return destination_table;
}


}  // namespace detail

std::unique_ptr<table> gather(table_view const& source_table, column_view const& gather_map,
			      bool check_bounds, rmm::mr::device_memory_resource* mr) {
  return detail::gather(source_table, gather_map, check_bounds, false, true, mr);
}

}  // namespace exp
}  // namespace cudf
