#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2018-2019, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "gather.hpp"
#include <cudf/copying.hpp>
#include <cudf/cudf.h>
#include <rmm/thrust_rmm_allocator.h>
#include <utilities/cudf_utils.h>
#include <utilities/type_dispatcher.hpp>
#include <bitmask/legacy/legacy_bitmask.hpp>
#include <cudf/table.hpp>
#include <string/nvcategory_util.hpp>

#include <algorithm>
#include <thrust/gather.h>

/**
 * @brief Operations for copying from one column to another
 * @file copying_ops.cu
 */

namespace {

/**---------------------------------------------------------------------------*
 * @brief Function object to check if an index is within the bounds [begin,
 * end).
 *
 *---------------------------------------------------------------------------**/
struct bounds_checker {
  gdf_index_type const begin;
  gdf_index_type const end;

  __device__ bounds_checker(gdf_index_type begin_, gdf_index_type end_)
      : begin{begin_}, end{end_} {}

  __device__ __forceinline__ bool operator()(gdf_index_type const index) {
    return ((index >= begin) && (index < end));
  }
};

/**---------------------------------------------------------------------------*
 * @brief Conditionally gathers the bits of a validity bitmask.
 *
 * Gathers the bits of a validity bitmask according to a gather map.
 * If `pred(stencil[i])` evaluates to true, then bit `i` in `destination_mask`
 * will equal bit `gather_map[i]` from the `source_mask`.
 *
 * If `pred(stencil[i])` evaluates to false, then bit `i` in `destination_mask`
 * will be set to 0.
 *
 * If any value appears in `gather_map` more than once, the result is undefined.
 *
 * If any of the range [source_mask, source_mask + num_source_rows) overlaps
 * [destination_mask, destination_mask + num_destination_rows), the result is
 * undefined.
 *
 * @tparam T The type of the stencil array
 * @tparam P The type of the predicate
 * @param[in] source_mask The mask whose bits will be gathered
 * @param[in] num_source_rows The number of bits in the source_mask
 * @param[out] destination_mask The output after gathering the input
 * @param[in] num_destination_rows The number of bits in the
 * destination_mask
 * @param[in] gather_map The map that indicates where elements from the
 * input will be gathered to in the output. Length must be equal to
 * `num_destination_rows`.
 * @param[in] stencil An array of values that will be evaluated by the
 * predicate. Length must be equal to `num_destination_rows`.
 * @param[in] pred Unary predicate applied to the stencil values
 *---------------------------------------------------------------------------**/
template <typename T, typename P>
__global__ void gather_bitmask_if_kernel(
    gdf_valid_type const* const __restrict__ source_mask,
    gdf_size_type const num_source_rows, gdf_valid_type* const destination_mask,
    gdf_size_type const num_destination_rows, gdf_index_type const* gather_map,
    T const* stencil, P pred) {
  using MaskType = uint32_t;
  constexpr uint32_t BITS_PER_MASK{sizeof(MaskType) * 8};

  // TODO: Update to use new bit_mask_t
  MaskType* const __restrict__ destination_mask32 =
      reinterpret_cast<MaskType*>(destination_mask);

  gdf_index_type destination_row = threadIdx.x + blockIdx.x * blockDim.x;

  auto active_threads =
      __ballot_sync(0xffffffff, destination_row < num_destination_rows);

  while (destination_row < num_destination_rows) {
    bool source_bit_is_valid{false};
    bool const predicate_is_true{pred(stencil[destination_row])};
    if (predicate_is_true) {
      // If the predicate for `destination_row` is false, it's valid for
      // `gather_map[destination_row]` to be out of bounds,
      // therefore, only use it if the predicate evaluates to true
      source_bit_is_valid =
          gdf_is_valid(source_mask, gather_map[destination_row]);
    }

    bool const destination_bit_is_valid{
        gdf_is_valid(destination_mask, destination_row)};

    // Use ballot to find all valid bits in this warp and create the output
    // bitmask element
    // If the predicate is false, and the destination bit was valid, don't
    // overwrite it
    MaskType const result_mask =
        __ballot_sync(active_threads,
                      (predicate_is_true and source_bit_is_valid) or
                          (not predicate_is_true and destination_bit_is_valid));

    gdf_index_type const output_element = destination_row / BITS_PER_MASK;

    // Only one thread writes output
    if (0 == threadIdx.x % warpSize) {
      destination_mask32[output_element] = result_mask;
    }

    destination_row += blockDim.x * gridDim.x;
    active_threads =
        __ballot_sync(active_threads, destination_row < num_destination_rows);
  }
}

/**---------------------------------------------------------------------------*
 * @brief Gathers the bits of a validity bitmask.
 *
 * Gathers the bits from the source bitmask into the destination bitmask
 * according to a `gather_map` such that bit `i` in `destination_mask` will be
 * equal to bit `gather_map[i]` from `source_bitmask`.
 *
 * Undefined behavior results if any value in `gather_map` is outside the range
 * [0, num_source_rows).
 *
 * If any value appears in `gather_map` more than once, the result is undefined.
 *
 * If any of the range [source_mask, source_mask + num_source_rows) overlaps
 * [destination_mask, destination_mask + num_destination_rows), the result is
 * undefined.
 *
 * @param[in] source_mask The mask whose bits will be gathered
 * @param[in] num_source_rows The number of bits in the source_mask
 * @param[out] destination_mask The output after gathering the input
 * @param[in] num_destination_rows The number of bits in the
 * destination_mask
 * @param[in] gather_map The map that indicates where elements from the
 * input will be gathered to in the output. Length must be equal to
 * `num_destination_rows`.
 *---------------------------------------------------------------------------**/
__global__ void gather_bitmask_kernel(
    gdf_valid_type const* const __restrict__ source_mask,
    gdf_size_type const num_source_rows, gdf_valid_type* const destination_mask,
    gdf_size_type const num_destination_rows,
    gdf_index_type const* __restrict__ gather_map) {
  using MaskType = uint32_t;
  constexpr uint32_t BITS_PER_MASK{sizeof(MaskType) * 8};

  // Cast bitmask to a type to a 4B type
  // TODO: Update to use new bit_mask_t
  MaskType* const __restrict__ destination_mask32 =
      reinterpret_cast<MaskType*>(destination_mask);

  gdf_index_type destination_row = threadIdx.x + blockIdx.x * blockDim.x;

  auto active_threads =
      __ballot_sync(0xffffffff, destination_row < num_destination_rows);

  while (destination_row < num_destination_rows) {
    bool const source_bit_is_valid{
        gdf_is_valid(source_mask, gather_map[destination_row])};

    // Use ballot to find all valid bits in this warp and create the output
    // bitmask element
    MaskType const result_mask{
        __ballot_sync(active_threads, source_bit_is_valid)};

    gdf_index_type const output_element = destination_row / BITS_PER_MASK;

    // Only one thread writes output
    if (0 == threadIdx.x % warpSize) {
      destination_mask32[output_element] = result_mask;
    }

    destination_row += blockDim.x * gridDim.x;
    active_threads =
        __ballot_sync(active_threads, destination_row < num_destination_rows);
  }
}

/**---------------------------------------------------------------------------*
 * @brief Gathers the bits from a source bitmask into a destination bitmask
 * based on a map.
 *
 * Gathers the bits from the source bitmask into the destination bitmask
 * according to a `gather_map` such that bit `i` in `destination_mask` will be
 * equal to bit `gather_map[i]` from `source_bitmask`.
 *
 * Optionally performs bounds checking on the values of the `gather_map` that
 * ignores values outside [0, num_source_rows). It is undefined behavior if a
 * value in `gather_map` is outside these bounds and bounds checking is not
 * enabled.
 *
 * If the same value appears more than once in `gather_map`, the result is
 * undefined.
 *
 * @param[in] source_mask The mask from which bits will be gathered
 * @param[in] num_source_rows The number of bits in the source_mask
 * @param[in,out] destination_mask The mask to which bits will be gathered.
 * Buffer must be preallocated with sufficient storage to hold
 * `num_destination_rows` bits.
 * @param[in] num_destination_rows The number of bits in the destionation_mask
 * @param[in] gather_map An array of indices that maps the bits in the source
 * bitmask to bits in the destination bitmask. The number of elements in the
 * `gather_map` must be equal to `num_destination_rows`.
 * @param[in] check_bounds Optionally perform bounds checking of values in
 * `gather_map`
 * @param[in] stream Optional CUDA stream on which to execute kernels
 *---------------------------------------------------------------------------**/
void gather_bitmask(gdf_valid_type const* source_mask,
                    gdf_size_type num_source_rows,
                    gdf_valid_type* destination_mask,
                    gdf_size_type num_destination_rows,
                    gdf_index_type const gather_map[],
                    bool check_bounds = false, hipStream_t stream = 0) {
  CUDF_EXPECTS(destination_mask != nullptr, "Missing valid buffer allocation");

  constexpr gdf_size_type BLOCK_SIZE{256};
  const gdf_size_type gather_grid_size =
      (num_destination_rows + BLOCK_SIZE - 1) / BLOCK_SIZE;

  gdf_valid_type* output_bitmask{destination_mask};

  // Allocate a temporary results buffer if gathering in-place
  bool const in_place{source_mask == destination_mask};
  rmm::device_vector<gdf_valid_type> temp_bitmask;
  if (in_place) {
    temp_bitmask.resize(gdf_valid_allocation_size(num_destination_rows));
    output_bitmask = temp_bitmask.data().get();
  }

  if (check_bounds) {
    gather_bitmask_if_kernel<<<gather_grid_size, BLOCK_SIZE, 0, stream>>>(
        source_mask, num_source_rows, output_bitmask, num_destination_rows,
        gather_map, gather_map, bounds_checker{0, num_source_rows});
  } else {
    gather_bitmask_kernel<<<gather_grid_size, BLOCK_SIZE, 0, stream>>>(
        source_mask, num_source_rows, output_bitmask, num_destination_rows,
        gather_map);
  }

  CHECK_STREAM(stream);

  if (in_place) {
    thrust::copy(rmm::exec_policy(stream)->on(stream), temp_bitmask.begin(),
                 temp_bitmask.end(), destination_mask);
  }

  CHECK_STREAM(stream);
}

/**---------------------------------------------------------------------------*
 * @brief Function object for gathering a type-erased
 * gdf_column. To be used with the cudf::type_dispatcher.
 *
 *---------------------------------------------------------------------------**/
struct column_gatherer {
  /**---------------------------------------------------------------------------*
   * @brief Type-dispatched function to gather from one column to another based
   * on a `gather_map`.
   *
   * @tparam ColumnType Dispatched type for the column being gathered
   * @param source_column The column to gather from
   * @param gather_map Array of indices that maps source elements to destination
   * elements
   * @param destination_column The column to gather into
   * @param check_bounds Optionally perform bounds checking on the values of
   * `gather_map`
   * @param stream Optional CUDA stream on which to execute kernels
   *---------------------------------------------------------------------------**/
  template <typename ColumnType>
  void operator()(gdf_column const* source_column,
                  gdf_index_type const gather_map[],
                  gdf_column* destination_column, bool check_bounds = false,
                  hipStream_t stream = 0) {
    ColumnType const* const source_data{
        static_cast<ColumnType const*>(source_column->data)};
    ColumnType* destination_data{
        static_cast<ColumnType*>(destination_column->data)};

    gdf_size_type const num_destination_rows{destination_column->size};

    // If gathering in-place, allocate temporary buffers to hold intermediate
    // results
    bool const in_place{source_data == destination_data};
    rmm::device_vector<ColumnType> temp_destination;
    if (in_place) {
      temp_destination.resize(num_destination_rows);
      destination_data = temp_destination.data().get();
    }

    if (check_bounds) {
      thrust::gather_if(rmm::exec_policy(stream)->on(stream), gather_map,
                        gather_map + num_destination_rows, gather_map,
                        source_data, destination_data,
                        bounds_checker{0, source_column->size});
    } else {
      thrust::gather(rmm::exec_policy(stream)->on(stream), gather_map,
                     gather_map + num_destination_rows, source_data,
                     destination_data);
    }

    // Copy temporary buffers used for in-place gather to destination column
    if (in_place) {
      thrust::copy(rmm::exec_policy(stream)->on(stream),
                   temp_destination.begin(), temp_destination.end(),
                   static_cast<ColumnType*>(destination_column->data));
    }

    if (destination_column->valid != nullptr) {
      gather_bitmask(source_column->valid, source_column->size,
                     destination_column->valid, num_destination_rows,
                     gather_map, check_bounds, stream);

      // TODO compute the null count in the gather_bitmask kernels
      set_null_count(*destination_column);
    }

    CHECK_STREAM(stream);
  }
};
}  // namespace

namespace cudf {
namespace detail {

void gather(table const* source_table, gdf_index_type const gather_map[],
            table* destination_table, bool check_bounds, hipStream_t stream) {
  CUDF_EXPECTS(nullptr != source_table, "source table is null");
  CUDF_EXPECTS(nullptr != destination_table, "destination table is null");

  // If the destination is empty, return immediately as there is nothing to
  // gather
  if (0 == destination_table->num_rows()) {
    return;
  }

  CUDF_EXPECTS(nullptr != gather_map, "gather_map is null");
  CUDF_EXPECTS(source_table->num_columns() == destination_table->num_columns(),
               "Mismatched number of columns");

  auto gather_column = [gather_map, check_bounds, stream](
                           gdf_column const* source, gdf_column* destination) {
    CUDF_EXPECTS(source->dtype == destination->dtype, "Column type mismatch");

    // If the source column has a valid buffer, the destination column must
    // also have one
    bool const source_has_nulls{source->valid != nullptr};
    bool const dest_has_nulls{destination->valid != nullptr};
    CUDF_EXPECTS((source_has_nulls && dest_has_nulls) || (not source_has_nulls),
                 "Missing destination validity buffer");

    // TODO: Each column could be gathered on a separate stream
    cudf::type_dispatcher(source->dtype, column_gatherer{}, source, gather_map,
                          destination, check_bounds, stream);

    return destination;
  };

  // Gather columns one-by-one
  std::transform(source_table->begin(), source_table->end(),
                 destination_table->begin(), destination_table->begin(),
                 gather_column);

}

}  // namespace detail

void gather(table const* source_table, gdf_index_type const gather_map[],
            table* destination_table) {
  detail::gather(source_table, gather_map, destination_table);
  nvcategory_gather_table(*source_table, *destination_table);
}

}  // namespace cudf
