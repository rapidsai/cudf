/*
 * Copyright (c) 2021, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <cudf/column/column_view.hpp>
#include <cudf/copying.hpp>
#include <cudf/detail/gather.cuh>
#include <cudf/detail/nvtx/ranges.hpp>
#include <cudf/table/table.hpp>
#include <cudf/table/table_view.hpp>

#include <rmm/cuda_stream_view.hpp>
#include <rmm/device_uvector.hpp>
#include <rmm/exec_policy.hpp>
#include <rmm/mr/device/per_device_resource.hpp>

#include <thrust/iterator/constant_iterator.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/iterator/transform_output_iterator.h>
#include <thrust/scan.h>

namespace cudf {
namespace detail {
std::unique_ptr<table> reverse(table_view const& source_table,
                               rmm::cuda_stream_view stream,
                               rmm::mr::device_memory_resource* mr)
{
  size_type num_rows = source_table.num_rows();
  auto elements =
    make_counting_transform_iterator(0, [num_rows] __device__(auto i) { return num_rows - i - 1; });
  auto elements_end = elements + source_table.num_rows();

  return gather(source_table, elements, elements_end, out_of_bounds_policy::DONT_CHECK, stream, mr);
}

std::unique_ptr<column> reverse(column_view const& source_column,
                                rmm::cuda_stream_view stream,
                                rmm::mr::device_memory_resource* mr)
{
  return std::move(cudf::reverse(table_view({source_column}))->release().front());
}
}  // namespace detail

std::unique_ptr<table> reverse(table_view const& source_table, rmm::mr::device_memory_resource* mr)
{
  CUDF_FUNC_RANGE();
  return detail::reverse(source_table, rmm::cuda_stream_default, mr);
}

std::unique_ptr<column> reverse(column_view const& source_column,
                                rmm::mr::device_memory_resource* mr)
{
  CUDF_FUNC_RANGE();
  return detail::reverse(source_column, rmm::cuda_stream_default, mr);
}
}  // namespace cudf
