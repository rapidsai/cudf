#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */
#include <cudf/copying.hpp>
#include <cudf/detail/copy.hpp>
#include <cudf/detail/scatter.cuh>
#include <cudf/detail/scatter.hpp>
#include <cudf/detail/gather.cuh>
#include <cudf/detail/gather.hpp>
#include <cudf/stream_compaction.hpp>
#include <cudf/detail/stream_compaction.hpp>
#include <cudf/utilities/traits.hpp>
#include <cudf/column/column_device_view.cuh>
#include <cudf/table/table_device_view.cuh>
#include <cudf/detail/utilities/cuda.cuh>
#include <cudf/strings/detail/scatter.cuh>
#include <cudf/strings/string_view.cuh>
#include "thrust/iterator/counting_iterator.h"
#include <numeric>

#include <thrust/iterator/discard_iterator.h>

namespace cudf {
namespace experimental {
namespace detail {

namespace {

struct dispatch_map_type {
    template <typename map_type, std::enable_if_t<std::is_integral<map_type>::value
     and not std::is_same<map_type, bool8>::value>* = nullptr>
    std::unique_ptr<table> operator()(
      table_view const& source, column_view const& scatter_map,
      table_view const& target, bool check_bounds,
      rmm::mr::device_memory_resource* mr, hipStream_t stream)
  {

    return detail::scatter<map_type>(source,
           scatter_map.begin<map_type>(),
           scatter_map.end<map_type>(),
           target, check_bounds, mr, stream);
  }

  template <typename map_type, std::enable_if_t<not std::is_integral<map_type>::value
      or std::is_same<map_type, bool8>::value>* = nullptr>
  std::unique_ptr<table> operator()(
      table_view const& source, column_view const& scatter_map,
      table_view const& target, bool check_bounds,
      rmm::mr::device_memory_resource* mr, hipStream_t stream)
  {
    CUDF_FAIL("Scatter map column must be an integral, non-boolean type");
  }

};

template <bool mark_true, typename MapIterator>
__global__ void marking_bitmask_kernel(
    mutable_column_device_view destination,
    MapIterator scatter_map,
    size_type num_scatter_rows)
{
  size_type row = threadIdx.x + blockIdx.x * blockDim.x;

  while (row < num_scatter_rows) {
    size_type const output_row = scatter_map[row];

    if (mark_true){
      destination.set_valid(output_row);
    } else {
      destination.set_null(output_row);
    }

    row += blockDim.x * gridDim.x;
  }
}

template <typename MapIterator>
void scatter_scalar_bitmask(std::vector<std::unique_ptr<scalar>> const& source,
    MapIterator scatter_map, size_type num_scatter_rows,
    std::vector<std::unique_ptr<column>>& target,
    rmm::mr::device_memory_resource* mr, hipStream_t stream)
{
  constexpr size_type block_size = 256;
  size_type const grid_size = grid_1d(num_scatter_rows, block_size).num_blocks;

  for (size_t i = 0; i < target.size(); ++i) {
    auto const source_is_valid = source[i]->is_valid(stream);
    if (target[i]->nullable() or not source_is_valid) {
      if (not target[i]->nullable()) {
        // Target must have a null mask if the source is not valid
        auto mask = create_null_mask(target[i]->size(), mask_state::ALL_VALID, stream, mr);
        target[i]->set_null_mask(std::move(mask), 0);
      }

      auto target_view = mutable_column_device_view::create(
        target[i]->mutable_view(), stream);

      auto bitmask_kernel = source_is_valid
        ? marking_bitmask_kernel<true, decltype(scatter_map)>
        : marking_bitmask_kernel<false, decltype(scatter_map)>;
      bitmask_kernel<<<grid_size, block_size, 0, stream>>>(
        *target_view, scatter_map, num_scatter_rows);
    }
  }
}

template <typename MapIterator>
struct column_scalar_scatterer {
  template <typename T, std::enable_if_t<is_fixed_width<T>()>* = nullptr>
  std::unique_ptr<column> operator()(std::unique_ptr<scalar> const& source,
      MapIterator scatter_iter, size_type scatter_rows, column_view const& target,
      rmm::mr::device_memory_resource* mr, hipStream_t stream)
  {
    auto result = std::make_unique<column>(target, stream, mr);
    auto result_view = result->mutable_view();

    // Use permutation iterator with constant index to dereference scalar data
    auto scalar_impl = static_cast<scalar_type_t<T>*>(source.get());
    auto scalar_iter = thrust::make_permutation_iterator(
      scalar_impl->data(), thrust::make_constant_iterator(0));

    thrust::scatter(rmm::exec_policy(stream)->on(stream), scalar_iter,
      scalar_iter + scatter_rows, scatter_iter,
      result_view.begin<T>());

    return result;
  }

  template <typename T, std::enable_if_t<not is_fixed_width<T>()>* = nullptr>
  std::unique_ptr<column> operator()(std::unique_ptr<scalar> const& source,
      MapIterator scatter_iter, size_type scatter_rows, column_view const& target,
      rmm::mr::device_memory_resource* mr, hipStream_t stream)
  {
    auto const scalar_impl = static_cast<string_scalar*>(source.get());
    auto const source_view = string_view(scalar_impl->data(), scalar_impl->size());
    auto const begin = thrust::make_constant_iterator(source_view);
    auto const end = begin + scatter_rows;
    return strings::detail::scatter(begin, end, scatter_iter, target, mr, stream);
  }
};

struct scatter_scalar_impl {
  template <typename T, std::enable_if_t<std::is_integral<T>::value
      and not std::is_same<T, bool8>::value>* = nullptr>
  std::unique_ptr<table> operator()(
      std::vector<std::unique_ptr<scalar>> const& source,
      column_view const& indices, table_view const& target, bool check_bounds,
      rmm::mr::device_memory_resource* mr, hipStream_t stream)
  {
    if (check_bounds) {
      auto const begin = -target.num_rows();
      auto const end = target.num_rows();
      auto bounds = bounds_checker<T>{begin, end};
      CUDF_EXPECTS(indices.size() == thrust::count_if(
        rmm::exec_policy(stream)->on(stream),
        indices.begin<T>(), indices.end<T>(), bounds),
        "Scatter map index out of bounds");
    }

    // Transform negative indices to index + target size
    auto scatter_rows = indices.size();
    auto scatter_iter = thrust::make_transform_iterator(
      indices.begin<T>(), index_converter<T>{target.num_rows()});

    // Second dispatch over data type per column
    auto result = std::vector<std::unique_ptr<column>>(target.num_columns());
    auto scatter_functor = column_scalar_scatterer<decltype(scatter_iter)>{};
    std::transform(source.begin(), source.end(), target.begin(), result.begin(),
      [=](auto const& source_scalar, auto const& target_col) {
        return type_dispatcher(source_scalar->type(), scatter_functor,
          source_scalar, scatter_iter, scatter_rows, target_col, mr, stream);
      });

    scatter_scalar_bitmask(source, scatter_iter, scatter_rows, result, mr, stream);

    return std::make_unique<table>(std::move(result));
  }

  template <typename T, std::enable_if_t<not std::is_integral<T>::value
      or std::is_same<T, bool8>::value>* = nullptr>
  std::unique_ptr<table> operator()(
      std::vector<std::unique_ptr<scalar>> const& source,
      column_view const& indices, table_view const& target, bool check_bounds,
      rmm::mr::device_memory_resource* mr, hipStream_t stream)
  {
    CUDF_FAIL("Scatter index column must be an integral, non-boolean type");
  }
};

struct scatter_to_tables_impl {
  template <typename T, std::enable_if_t<std::is_integral<T>::value
      and not std::is_same<T, bool8>::value>* = nullptr>
  std::vector<std::unique_ptr<table>> operator()(
      table_view const& input, column_view const& partition_map,
      rmm::mr::device_memory_resource* mr, hipStream_t stream)
  {
    // Make a mutable copy of the partition map
    auto d_partitions = rmm::device_vector<T>(
      partition_map.begin<T>(), partition_map.end<T>());

    // Initialize gather maps and offsets to sequence
    auto d_gather_maps = rmm::device_vector<size_type>(partition_map.size());
    auto d_offsets = rmm::device_vector<size_type>(partition_map.size());
    thrust::sequence(rmm::exec_policy(stream)->on(stream),
      d_gather_maps.begin(), d_gather_maps.end());
    thrust::sequence(rmm::exec_policy(stream)->on(stream),
      d_offsets.begin(), d_offsets.end());

    // Sort sequence using partition map as key to generate gather maps
    thrust::stable_sort_by_key(rmm::exec_policy(stream)->on(stream),
      d_partitions.begin(), d_partitions.end(), d_gather_maps.begin());

    // Reduce unique partitions to extract gather map offsets from sequence
    auto end = thrust::unique_by_key(rmm::exec_policy(stream)->on(stream),
      d_partitions.begin(), d_partitions.end(), d_offsets.begin());

    // Copy partition indices and gather map offsets to host
    auto partitions = thrust::host_vector<T>(d_partitions.begin(), end.first);
    auto offsets = thrust::host_vector<size_type>(d_offsets.begin(), end.second);
    offsets.push_back(partition_map.size());

    CUDF_EXPECTS(partitions.front() >= 0, "Invalid negative partition index");
    auto output = std::vector<std::unique_ptr<table>>(partitions.back() + 1);

    size_t next_partition = 0;
    for (size_t index = 0; index < partitions.size(); ++index) {
      auto const partition = static_cast<size_t>(partitions[index]);

      // Create empty tables for unused partitions
      for (; next_partition < partition; ++next_partition) {
        output[next_partition] = empty_like(input);
      }

      // Gather input rows for the current partition (second dispatch for column types)
      auto const data = d_gather_maps.data().get() + offsets[index];
      auto const size = offsets[index + 1] - offsets[index];
      auto const gather_map = column_view(data_type(INT32), size, data);
      output[partition] = gather(input, gather_map, false, false, false, mr, stream);

      next_partition = partition + 1;
    }

    return output;
  }

  template <typename T, std::enable_if_t<not std::is_integral<T>::value
      or std::is_same<T, bool8>::value>* = nullptr>
  std::vector<std::unique_ptr<table>> operator()(
      table_view const& input, column_view const& partition_map,
      rmm::mr::device_memory_resource* mr, hipStream_t stream)
  {
    CUDF_FAIL("Partition map column must be an integral, non-boolean type");
  }
};

}  // namespace

std::unique_ptr<table> scatter(
    table_view const& source, column_view const& scatter_map,
    table_view const& target, bool check_bounds,
    rmm::mr::device_memory_resource* mr,
    hipStream_t stream)
{
  CUDF_EXPECTS(source.num_columns() == target.num_columns(),
    "Number of columns in source and target not equal");
  CUDF_EXPECTS(scatter_map.size() <= source.num_rows(),
    "Size of scatter map must be equal to or less than source rows");
  CUDF_EXPECTS(std::equal(source.begin(), source.end(), target.begin(),
    [](auto const& col1, auto const& col2) {
      return col1.type().id() == col2.type().id();
    }), "Column types do not match between source and target");
  CUDF_EXPECTS(scatter_map.has_nulls() == false, "Scatter map contains nulls");

  if (scatter_map.size() == 0) {
    return std::make_unique<table>(target, stream, mr);
  }

  // First dispatch for scatter map index type
  return type_dispatcher(scatter_map.type(), dispatch_map_type{}, source,
    scatter_map, target, check_bounds, mr, stream);
}

std::unique_ptr<table> scatter(
    std::vector<std::unique_ptr<scalar>> const& source, column_view const& indices,
    table_view const& target, bool check_bounds,
    rmm::mr::device_memory_resource* mr,
    hipStream_t stream)
{
  CUDF_EXPECTS(source.size() == static_cast<size_t>(target.num_columns()),
    "Number of columns in source and target not equal");
  CUDF_EXPECTS(std::equal(source.begin(), source.end(), target.begin(),
    [](auto const& scalar, auto const& col) {
      return scalar->type().id() == col.type().id();
    }), "Column types do not match between source and target");
  CUDF_EXPECTS(indices.has_nulls() == false, "indices contains nulls");

  if (indices.size() == 0) {
    return std::make_unique<table>(target, stream, mr);
  }

  // First dispatch for scatter index type
  return type_dispatcher(indices.type(), scatter_scalar_impl{}, source,
    indices, target, check_bounds, mr, stream);
}

std::vector<std::unique_ptr<table>> scatter_to_tables(
    table_view const& input, column_view const& partition_map,
    rmm::mr::device_memory_resource* mr,
    hipStream_t stream)
{
  CUDF_EXPECTS(partition_map.size() <= input.num_rows(), "scatter map larger than input");
  CUDF_EXPECTS(partition_map.has_nulls() == false, "scatter map contains nulls");

  if (partition_map.size() == 0 || input.num_rows() == 0) {
    return std::vector<std::unique_ptr<table>>{};
  }

  // First dispatch for scatter index type
  return type_dispatcher(partition_map.type(), scatter_to_tables_impl{},
    input, partition_map, mr, stream);
}

std::unique_ptr<column> boolean_mask_scatter(
    column_view const& input, column_view const& target,
    column_view const& boolean_mask,
    rmm::mr::device_memory_resource *mr,
    hipStream_t stream) {

    auto indices = cudf::make_numeric_column(data_type{INT32},
                                  target.size(), mask_state::UNALLOCATED, stream, mr);
    auto mutable_indices = indices->mutable_view();

    thrust::sequence(rmm::exec_policy(stream)->on(stream),
                     mutable_indices.begin<size_type>(),
                     mutable_indices.end<size_type>(),
                     0);

    // The scatter map is actually a table with only one column, which is scatter map.
    auto scatter_map = detail::apply_boolean_mask(table_view{{indices->view()}}, boolean_mask,
                                                  mr, stream);
    auto output_table =  detail::scatter(table_view{{input}}, scatter_map->get_column(0).view(),
                                             table_view{{target}}, false, mr, stream);

    // There is only one column in output_table
    return std::make_unique<column>(std::move(output_table->get_column(0)));
}

std::unique_ptr<column> boolean_mask_scatter(
    scalar const& input, column_view const& target,
    column_view const& boolean_mask,
    rmm::mr::device_memory_resource *mr,
    hipStream_t stream) {
  
    return detail::copy_if_else(input, target, boolean_mask, mr, stream);
}

std::unique_ptr<table> boolean_mask_scatter(
    table_view const& input, table_view const& target,
    column_view const& boolean_mask,
    rmm::mr::device_memory_resource *mr,
    hipStream_t stream) {

    CUDF_EXPECTS(input.num_columns() == target.num_columns(), "Mismatch in number of input columns and target columns");
    CUDF_EXPECTS(boolean_mask.size() == target.num_rows(), "Boolean mask size and number of target rows mismatch");
    CUDF_EXPECTS(boolean_mask.type().id() == BOOL8, "Mask must be of Boolean type");
    // Count valid pair of input and columns as per type at each column index i
    CUDF_EXPECTS(std::all_of(thrust::counting_iterator<size_type>(0),
                                        thrust::counting_iterator<size_type>(target.num_columns()),
                                        [&input, &target](auto index){
                                            return ((input.column(index).type().id()) == (target.column(index).type().id()));
                                        }), "Type mismatch in input column and target column");

    if (target.num_rows() != 0) {
        std::vector<std::unique_ptr<column>> out_columns(target.num_columns());
        std::transform(input.begin(), input.end(), target.begin(),
                out_columns.begin(),
                [&boolean_mask, mr, stream] (auto const& input_column, auto const& target_column){
                return boolean_mask_scatter(input_column, target_column,
                        boolean_mask, mr, stream);});

        return std::make_unique<experimental::table>(std::move(out_columns));
    } else {
        return experimental::empty_like(target);
    }
}

std::unique_ptr<table> boolean_mask_scatter(
    std::vector<std::reference_wrapper<scalar>> const& input,
    table_view const& target,
    column_view const& boolean_mask,
    rmm::mr::device_memory_resource *mr,
    hipStream_t stream) {

    CUDF_EXPECTS(static_cast<size_type>(input.size()) == target.num_columns(), "Mismatch in number of scalars and target columns");
    CUDF_EXPECTS(boolean_mask.size() == target.num_rows(), "Boolean mask size and number of target rows mismatch");
    CUDF_EXPECTS(boolean_mask.type().id() == BOOL8, "Mask must be of Boolean type");

    // Count valid pair of input and columns as per type at each column/scalar index i
    CUDF_EXPECTS(std::all_of(thrust::counting_iterator<size_type>(0),
                                        thrust::counting_iterator<size_type>(target.num_columns()),
                                        [&input, &target](auto index){
                                            return (input[index].get().type().id() == target.column(index).type().id());
                                        }), "Type mismatch in input scalar and target column");

    if (target.num_rows() != 0) {
        std::vector<std::unique_ptr<column>> out_columns(target.num_columns());
        std::transform(input.begin(), input.end(), target.begin(),
                out_columns.begin(),
                [&boolean_mask, mr, stream] (auto const& scalar, auto const& target_column){
                return boolean_mask_scatter(scalar.get(), target_column,
                        boolean_mask, mr, stream);});

        return std::make_unique<experimental::table>(std::move(out_columns));
    } else {
        return experimental::empty_like(target);
    }
}

}  // namespace detail

std::unique_ptr<table> scatter(
    table_view const& source, column_view const& scatter_map,
    table_view const& target, bool check_bounds,
    rmm::mr::device_memory_resource* mr) {
  return detail::scatter(source, scatter_map, target, check_bounds, mr);
}

std::unique_ptr<table> scatter(
    std::vector<std::unique_ptr<scalar>> const& source, column_view const& indices,
    table_view const& target, bool check_bounds,
    rmm::mr::device_memory_resource* mr) {
  return detail::scatter(source, indices, target, check_bounds, mr);
}

std::vector<std::unique_ptr<table>> scatter_to_tables(
    table_view const& input, column_view const& partition_map,
    rmm::mr::device_memory_resource* mr) {
  return detail::scatter_to_tables(input, partition_map, mr);
}

std::unique_ptr<table> boolean_mask_scatter(
    table_view const& input, table_view const& target,
    column_view const& boolean_mask,
    rmm::mr::device_memory_resource *mr) {
    return detail::boolean_mask_scatter(input, target, boolean_mask, mr);
}

std::unique_ptr<table> boolean_mask_scatter(
    std::vector<std::reference_wrapper<scalar>> const& input, table_view const& target,
    column_view const& boolean_mask,
    rmm::mr::device_memory_resource *mr) {
    return detail::boolean_mask_scatter(input, target, boolean_mask, mr);
}

}  // namespace experimental
}  // namespace cudf
