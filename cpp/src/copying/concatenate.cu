#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2020, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */
#include <cudf/detail/concatenate.cuh>

#include <cudf/column/column.hpp>
#include <cudf/column/column_device_view.cuh>
#include <cudf/detail/copy.hpp>
#include <cudf/detail/nvtx/ranges.hpp>
#include <cudf/detail/utilities/cuda.cuh>
#include <cudf/dictionary/detail/concatenate.hpp>
#include <cudf/lists/detail/concatenate.hpp>
#include <cudf/strings/detail/concatenate.hpp>
#include <cudf/table/table.hpp>
#include <cudf/table/table_device_view.cuh>

#include <thrust/binary_search.h>
#include <thrust/transform_scan.h>

#include <algorithm>
#include <numeric>
#include <utility>

namespace cudf {
namespace detail {
// From benchmark data, the fused kernel optimization appears to perform better
// when there are more than a trivial number of columns, or when the null mask
// can also be computed at the same time
constexpr bool use_fused_kernel_heuristic(bool const has_nulls, size_t const num_columns)
{
  return has_nulls || num_columns > 4;
}

auto create_device_views(std::vector<column_view> const& views, hipStream_t stream)
{
  // Create device views for each input view
  using CDViewPtr =
    decltype(column_device_view::create(std::declval<column_view>(), std::declval<hipStream_t>()));
  auto device_view_owners = std::vector<CDViewPtr>(views.size());
  std::transform(
    views.cbegin(), views.cend(), device_view_owners.begin(), [stream](auto const& col) {
      // TODO creating this device view can invoke null count computation
      // even though it isn't used. See this issue:
      // https://github.com/rapidsai/cudf/issues/4368
      return column_device_view::create(col, stream);
    });

  // Assemble contiguous array of device views
  auto device_views = thrust::host_vector<column_device_view>();
  device_views.reserve(views.size());
  std::transform(device_view_owners.cbegin(),
                 device_view_owners.cend(),
                 std::back_inserter(device_views),
                 [](auto const& col) { return *col; });
  // TODO each of these device vector copies invoke stream synchronization
  // which appears to add unnecessary overhead. See this issue:
  // https://github.com/rapidsai/rmm/issues/120
  auto d_views = rmm::device_vector<column_device_view>{device_views};

  // Compute the partition offsets
  auto offsets = thrust::host_vector<size_t>(views.size() + 1);
  thrust::transform_inclusive_scan(
    thrust::host,
    device_views.cbegin(),
    device_views.cend(),
    std::next(offsets.begin()),
    [](auto const& col) { return col.size(); },
    thrust::plus<size_t>{});
  auto const d_offsets   = rmm::device_vector<size_t>{offsets};
  auto const output_size = offsets.back();

  return std::make_tuple(
    std::move(device_view_owners), std::move(d_views), std::move(d_offsets), output_size);
}

/**
 * @brief Concatenates the null mask bits of all the column device views in the
 * `views` array to the destination bitmask.
 *
 * @param views Array of column_device_view
 * @param output_offsets Prefix sum of sizes of elements of `views`
 * @param number_of_views Size of `views` array
 * @param dest_mask The output buffer to copy null masks into
 * @param number_of_mask_bits The total number of null masks bits that are being
 * copied
 **/
__global__ void concatenate_masks_kernel(column_device_view const* views,
                                         size_t const* output_offsets,
                                         size_type number_of_views,
                                         bitmask_type* dest_mask,
                                         size_type number_of_mask_bits)
{
  size_type mask_index = threadIdx.x + blockIdx.x * blockDim.x;

  auto active_mask = __ballot_sync(0xFFFF'FFFF, mask_index < number_of_mask_bits);

  while (mask_index < number_of_mask_bits) {
    size_type const source_view_index =
      thrust::upper_bound(
        thrust::seq, output_offsets, output_offsets + number_of_views, mask_index) -
      output_offsets - 1;
    bool bit_is_set = 1;
    if (source_view_index < number_of_views) {
      size_type const column_element_index = mask_index - output_offsets[source_view_index];
      bit_is_set = views[source_view_index].is_valid(column_element_index);
    }
    bitmask_type const new_word = __ballot_sync(active_mask, bit_is_set);

    if (threadIdx.x % detail::warp_size == 0) { dest_mask[word_index(mask_index)] = new_word; }

    mask_index += blockDim.x * gridDim.x;
    active_mask = __ballot_sync(active_mask, mask_index < number_of_mask_bits);
  }
}

void concatenate_masks(rmm::device_vector<column_device_view> const& d_views,
                       rmm::device_vector<size_t> const& d_offsets,
                       bitmask_type* dest_mask,
                       size_type output_size,
                       hipStream_t stream)
{
  constexpr size_type block_size{256};
  cudf::detail::grid_1d config(output_size, block_size);
  concatenate_masks_kernel<<<config.num_blocks, config.num_threads_per_block, 0, stream>>>(
    d_views.data().get(),
    d_offsets.data().get(),
    static_cast<size_type>(d_views.size()),
    dest_mask,
    output_size);
}

void concatenate_masks(std::vector<column_view> const& views,
                       bitmask_type* dest_mask,
                       hipStream_t stream)
{
  // Preprocess and upload inputs to device memory
  auto const device_views = create_device_views(views, stream);
  auto const& d_views     = std::get<1>(device_views);
  auto const& d_offsets   = std::get<2>(device_views);
  auto const output_size  = std::get<3>(device_views);

  concatenate_masks(d_views, d_offsets, dest_mask, output_size, stream);
}

template <typename T, size_type block_size, bool Nullable>
__global__ void fused_concatenate_kernel(column_device_view const* input_views,
                                         size_t const* input_offsets,
                                         size_type num_input_views,
                                         mutable_column_device_view output_view,
                                         size_type* out_valid_count)
{
  auto const output_size = output_view.size();
  auto* output_data      = output_view.data<T>();

  size_type output_index     = threadIdx.x + blockIdx.x * blockDim.x;
  size_type warp_valid_count = 0;

  unsigned active_mask;
  if (Nullable) { active_mask = __ballot_sync(0xFFFF'FFFF, output_index < output_size); }
  while (output_index < output_size) {
    // Lookup input index by searching for output index in offsets
    // thrust::prev isn't in CUDA 10.0, so subtracting 1 here instead
    auto const offset_it =
      -1 + thrust::upper_bound(
             thrust::seq, input_offsets, input_offsets + num_input_views, output_index);
    size_type const partition_index = offset_it - input_offsets;

    // Copy input data to output
    auto const offset_index   = output_index - *offset_it;
    auto const& input_view    = input_views[partition_index];
    auto const* input_data    = input_view.data<T>();
    output_data[output_index] = input_data[offset_index];

    if (Nullable) {
      bool const bit_is_set       = input_view.is_valid(offset_index);
      bitmask_type const new_word = __ballot_sync(active_mask, bit_is_set);

      // First thread writes bitmask word
      if (threadIdx.x % detail::warp_size == 0) {
        output_view.null_mask()[word_index(output_index)] = new_word;
      }

      warp_valid_count += __popc(new_word);
    }

    output_index += blockDim.x * gridDim.x;
    if (Nullable) { active_mask = __ballot_sync(active_mask, output_index < output_size); }
  }

  if (Nullable) {
    using detail::single_lane_block_sum_reduce;
    auto block_valid_count = single_lane_block_sum_reduce<block_size, 0>(warp_valid_count);
    if (threadIdx.x == 0) { atomicAdd(out_valid_count, block_valid_count); }
  }
}

template <typename T>
std::unique_ptr<column> fused_concatenate(std::vector<column_view> const& views,
                                          bool const has_nulls,
                                          rmm::mr::device_memory_resource* mr,
                                          hipStream_t stream)
{
  using mask_policy = cudf::mask_allocation_policy;

  // Preprocess and upload inputs to device memory
  auto const device_views = create_device_views(views, stream);
  auto const& d_views     = std::get<1>(device_views);
  auto const& d_offsets   = std::get<2>(device_views);
  auto const output_size  = std::get<3>(device_views);

  CUDF_EXPECTS(output_size < std::numeric_limits<size_type>::max(),
               "Total number of concatenated rows exceeds size_type range");

  // Allocate output
  auto const policy = has_nulls ? mask_policy::ALWAYS : mask_policy::NEVER;
  auto out_col      = detail::allocate_like(views.front(), output_size, policy, mr, stream);
  out_col->set_null_count(0);  // prevent null count from being materialized
  auto out_view   = out_col->mutable_view();
  auto d_out_view = mutable_column_device_view::create(out_view, stream);

  rmm::device_scalar<size_type> d_valid_count(0, stream);

  // Launch kernel
  constexpr size_type block_size{256};
  cudf::detail::grid_1d config(output_size, block_size);
  auto const kernel = has_nulls ? fused_concatenate_kernel<T, block_size, true>
                                : fused_concatenate_kernel<T, block_size, false>;
  kernel<<<config.num_blocks, config.num_threads_per_block, 0, stream>>>(
    d_views.data().get(),
    d_offsets.data().get(),
    static_cast<size_type>(d_views.size()),
    *d_out_view,
    d_valid_count.data());

  if (has_nulls) { out_col->set_null_count(output_size - d_valid_count.value(stream)); }

  return out_col;
}

template <typename T>
std::unique_ptr<column> for_each_concatenate(std::vector<column_view> const& views,
                                             bool const has_nulls,
                                             rmm::mr::device_memory_resource* mr,
                                             hipStream_t stream)
{
  size_type const total_element_count =
    std::accumulate(views.begin(), views.end(), 0, [](auto accumulator, auto const& v) {
      return accumulator + v.size();
    });

  using mask_policy = cudf::mask_allocation_policy;
  auto const policy = has_nulls ? mask_policy::ALWAYS : mask_policy::NEVER;
  auto col          = cudf::allocate_like(views.front(), total_element_count, policy, mr);

  col->set_null_count(0);             // prevent null count from being materialized...
  auto m_view = col->mutable_view();  // ...when we take a mutable view

  auto count = 0;
  for (auto& v : views) {
    thrust::copy(
      rmm::exec_policy()->on(stream), v.begin<T>(), v.end<T>(), m_view.begin<T>() + count);
    count += v.size();
  }

  // If concatenated column is nullable, proceed to calculate it
  if (has_nulls) {
    cudf::detail::concatenate_masks(views, (col->mutable_view()).null_mask(), stream);
  }

  return col;
}

struct concatenate_dispatch {
  std::vector<column_view> const& views;
  rmm::mr::device_memory_resource* mr;
  hipStream_t stream;

  // fixed width
  template <typename T>
  std::unique_ptr<column> operator()()
  {
    bool const has_nulls =
      std::any_of(views.cbegin(), views.cend(), [](auto const& col) { return col.has_nulls(); });

    using Type = device_storage_type_t<T>;

    // Use a heuristic to guess when the fused kernel will be faster
    if (use_fused_kernel_heuristic(has_nulls, views.size())) {
      return fused_concatenate<Type>(views, has_nulls, mr, stream);
    } else {
      return for_each_concatenate<Type>(views, has_nulls, mr, stream);
    }
  }
};

template <>
std::unique_ptr<column> concatenate_dispatch::operator()<cudf::dictionary32>()
{
  return cudf::dictionary::detail::concatenate(views, stream, mr);
}

template <>
std::unique_ptr<column> concatenate_dispatch::operator()<cudf::string_view>()
{
  return cudf::strings::detail::concatenate(views, mr, stream);
}

template <>
std::unique_ptr<column> concatenate_dispatch::operator()<cudf::list_view>()
{
  return cudf::lists::detail::concatenate(views, stream, mr);
}

// Concatenates the elements from a vector of column_views
std::unique_ptr<column> concatenate(std::vector<column_view> const& columns_to_concat,
                                    rmm::mr::device_memory_resource* mr,
                                    hipStream_t stream)
{
  CUDF_EXPECTS(not columns_to_concat.empty(), "Unexpected empty list of columns to concatenate.");

  data_type const type = columns_to_concat.front().type();
  CUDF_EXPECTS(std::all_of(columns_to_concat.begin(),
                           columns_to_concat.end(),
                           [&type](auto const& c) { return c.type() == type; }),
               "Type mismatch in columns to concatenate.");

  if (std::all_of(columns_to_concat.begin(), columns_to_concat.end(), [](column_view const& c) {
        return c.is_empty();
      })) {
    return empty_like(columns_to_concat.front());
  }

  return type_dispatcher(type, concatenate_dispatch{columns_to_concat, mr, stream});
}

std::unique_ptr<table> concatenate(std::vector<table_view> const& tables_to_concat,
                                   rmm::mr::device_memory_resource* mr,
                                   hipStream_t stream)
{
  if (tables_to_concat.empty()) { return std::make_unique<table>(); }

  table_view const first_table = tables_to_concat.front();
  CUDF_EXPECTS(std::all_of(tables_to_concat.cbegin(),
                           tables_to_concat.cend(),
                           [&first_table](auto const& t) {
                             return t.num_columns() == first_table.num_columns() &&
                                    have_same_types(first_table, t);
                           }),
               "Mismatch in table columns to concatenate.");

  std::vector<std::unique_ptr<column>> concat_columns;
  for (size_type i = 0; i < first_table.num_columns(); ++i) {
    std::vector<column_view> cols;
    std::transform(tables_to_concat.cbegin(),
                   tables_to_concat.cend(),
                   std::back_inserter(cols),
                   [i](auto const& t) { return t.column(i); });
    concat_columns.emplace_back(detail::concatenate(cols, mr, stream));
  }
  return std::make_unique<table>(std::move(concat_columns));
}

}  // namespace detail

rmm::device_buffer concatenate_masks(std::vector<column_view> const& views,
                                     rmm::mr::device_memory_resource* mr)
{
  bool const has_nulls =
    std::any_of(views.begin(), views.end(), [](const column_view col) { return col.has_nulls(); });
  if (has_nulls) {
    size_type const total_element_count =
      std::accumulate(views.begin(), views.end(), 0, [](auto accumulator, auto const& v) {
        return accumulator + v.size();
      });

    rmm::device_buffer null_mask =
      create_null_mask(total_element_count, mask_state::UNINITIALIZED, 0, mr);

    detail::concatenate_masks(views, static_cast<bitmask_type*>(null_mask.data()), 0);

    return null_mask;
  }
  // no nulls, so return an empty device buffer
  return rmm::device_buffer{0, (hipStream_t)0, mr};
}

// Concatenates the elements from a vector of column_views
std::unique_ptr<column> concatenate(std::vector<column_view> const& columns_to_concat,
                                    rmm::mr::device_memory_resource* mr)
{
  CUDF_FUNC_RANGE();
  return detail::concatenate(columns_to_concat, mr, 0);
}

std::unique_ptr<table> concatenate(std::vector<table_view> const& tables_to_concat,
                                   rmm::mr::device_memory_resource* mr)
{
  CUDF_FUNC_RANGE();
  return detail::concatenate(tables_to_concat, mr, 0);
}

}  // namespace cudf
