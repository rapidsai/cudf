/*
 * Copyright (c) 2022, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <cudf/column/column.hpp>
#include <cudf/column/column_factories.hpp>
#include <cudf/detail/structs/utilities.hpp>
#include <cudf/detail/utilities/linked_column.hpp>
#include <cudf/detail/utilities/vector_factories.hpp>
#include <cudf/lists/lists_column_view.hpp>
#include <cudf/table/experimental/row_operators.cuh>
#include <cudf/table/table_view.hpp>
#include <cudf/utilities/type_checks.hpp>

#include <jit/type.hpp>

#include <thrust/iterator/transform_iterator.h>

namespace cudf {
namespace experimental {

namespace {

/**
 * @brief Removes the offsets of struct column's children
 *
 * @param c The column whose children are to be un-sliced
 * @return Children of `c` with offsets removed
 */
std::vector<column_view> unslice_children(column_view const& c)
{
  if (c.type().id() == type_id::STRUCT) {
    auto child_it = thrust::make_transform_iterator(c.child_begin(), [](auto const& child) {
      return column_view(
        child.type(),
        child.offset() + child.size(),  // This is hacky, we don't know the actual unsliced size but
                                        // it is at least offset + size
        child.head(),
        child.null_mask(),
        child.null_count(),
        0,
        unslice_children(child));
    });
    return {child_it, child_it + c.num_children()};
  }
  return {c.child_begin(), c.child_end()};
};

/**
 * @brief Removes the child column offsets of struct columns in a table.
 *
 * Given a table, this replaces any struct columns with similar struct columns that have their
 * offsets removed from their children. Structs that are children of list columns are not affected.
 *
 */
table_view remove_struct_child_offsets(table_view table)
{
  std::vector<column_view> cols;
  cols.reserve(table.num_columns());
  std::transform(table.begin(), table.end(), std::back_inserter(cols), [&](column_view const& c) {
    return column_view(c.type(),
                       c.size(),
                       c.head<uint8_t>(),
                       c.null_mask(),
                       c.null_count(),
                       c.offset(),
                       unslice_children(c));
  });
  return table_view(cols);
}

/**
 * @brief Decompose all struct columns in a table
 *
 * If a struct column is a tree with N leaves, then this function decomposes the tree into
 * N "linear trees" (branch factor == 1) and prunes common parents. Also returns a vector of
 * per-column `depth`s.
 *
 * A `depth` value is the number of nested levels as parent of the column in the original,
 * non-decomposed table, which are pruned during decomposition.
 *
 * For example, if the original table has a column `Struct<Struct<int, float>, decimal>`,
 *
 *      S1
 *     / \
 *    S2  d
 *   / \
 *  i   f
 *
 * then after decomposition, we get three columns:
 * `Struct<Struct<int>>`, `float`, and `decimal`.
 *
 *  0   2   1  <- depths
 *  S1
 *  |
 *  S2      d
 *  |
 *  i   f
 *
 * The depth of the first column is 0 because it contains all its parent levels, while the depth
 * of the second column is 2 because two of its parent struct levels were pruned.
 *
 * Similarly, a struct column of type Struct<int, Struct<float, decimal>> is decomposed as follows
 *
 *     S1
 *    / \
 *   i   S2
 *      / \
 *     f   d
 *
 *  0   1   2  <- depths
 *  S1  S2  d
 *  |   |
 *  i   f
 *
 * When list columns are present, the decomposition is performed similarly to pure structs but list
 * parent columns are NOT pruned
 *
 * For example, if the original table has a column `List<Struct<int, float>>`,
 *
 *    L
 *    |
 *    S
 *   / \
 *  i   f
 *
 * after decomposition, we get two columns
 *
 *  L   L
 *  |   |
 *  S   f
 *  |
 *  i
 *
 * The list parents are still needed to define the range of elements in the leaf that belong to the
 * same row.
 *
 * @param table The table whose struct columns to decompose.
 * @param column_order The per-column order if using output with lexicographic comparison
 * @param null_precedence The per-column null precedence
 * @return A tuple containing a table with all struct columns decomposed, new corresponding column
 *         orders and null precedences and depths of the linearized branches
 */
auto decompose_structs(table_view table,
                       host_span<order const> column_order         = {},
                       host_span<null_order const> null_precedence = {})
{
  auto linked_columns = detail::table_to_linked_columns(table);

  std::vector<column_view> verticalized_columns;
  std::vector<order> new_column_order;
  std::vector<null_order> new_null_precedence;
  std::vector<int> verticalized_col_depths;
  for (size_t col_idx = 0; col_idx < linked_columns.size(); ++col_idx) {
    detail::linked_column_view const* col = linked_columns[col_idx].get();
    if (is_nested(col->type())) {
      // convert and insert
      std::vector<std::vector<detail::linked_column_view const*>> flattened;
      std::function<void(
        detail::linked_column_view const*, std::vector<detail::linked_column_view const*>*, int)>
        recursive_child = [&](detail::linked_column_view const* c,
                              std::vector<detail::linked_column_view const*>* branch,
                              int depth) {
          branch->push_back(c);
          if (c->type().id() == type_id::LIST) {
            recursive_child(
              c->children[lists_column_view::child_column_index].get(), branch, depth + 1);
          } else if (c->type().id() == type_id::STRUCT) {
            for (size_t child_idx = 0; child_idx < c->children.size(); ++child_idx) {
              if (child_idx > 0) {
                verticalized_col_depths.push_back(depth + 1);
                branch = &flattened.emplace_back();
              }
              recursive_child(c->children[child_idx].get(), branch, depth + 1);
            }
          }
        };
      auto& branch = flattened.emplace_back();
      verticalized_col_depths.push_back(0);
      recursive_child(col, &branch, 0);

      for (auto const& branch : flattened) {
        column_view temp_col = *branch.back();
        for (auto it = branch.crbegin() + 1; it < branch.crend(); ++it) {
          auto const& prev_col = *(*it);
          auto children =
            (prev_col.type().id() == type_id::LIST)
              ? std::vector<column_view>{*prev_col
                                            .children[lists_column_view::offsets_column_index],
                                         temp_col}
              : std::vector<column_view>{temp_col};
          temp_col = column_view(prev_col.type(),
                                 prev_col.size(),
                                 nullptr,
                                 prev_col.null_mask(),
                                 UNKNOWN_NULL_COUNT,
                                 prev_col.offset(),
                                 std::move(children));
        }
        // Traverse upward and include any list columns in the ancestors
        for (detail::linked_column_view* parent = branch.front()->parent; parent;
             parent                             = parent->parent) {
          if (parent->type().id() == type_id::LIST) {
            // Include this parent
            temp_col = column_view(
              parent->type(),
              parent->size(),
              nullptr,  // list has no data of its own
              nullptr,  // If we're going through this then nullmask is already in another branch
              UNKNOWN_NULL_COUNT,
              parent->offset(),
              {*parent->children[lists_column_view::offsets_column_index], temp_col});
          } else if (parent->type().id() == type_id::STRUCT) {
            // Replace offset with parent's offset
            temp_col = column_view(temp_col.type(),
                                   parent->size(),
                                   temp_col.head(),
                                   temp_col.null_mask(),
                                   UNKNOWN_NULL_COUNT,
                                   parent->offset(),
                                   {temp_col.child_begin(), temp_col.child_end()});
          }
        }
        verticalized_columns.push_back(temp_col);
      }
      if (not column_order.empty()) {
        new_column_order.insert(new_column_order.end(), flattened.size(), column_order[col_idx]);
      }
      if (not null_precedence.empty()) {
        new_null_precedence.insert(
          new_null_precedence.end(), flattened.size(), null_precedence[col_idx]);
      }
    } else {
      verticalized_columns.push_back(*col);
      verticalized_col_depths.push_back(0);
      if (not column_order.empty()) { new_column_order.push_back(column_order[col_idx]); }
      if (not null_precedence.empty()) { new_null_precedence.push_back(null_precedence[col_idx]); }
    }
  }
  return std::make_tuple(table_view(verticalized_columns),
                         std::move(new_column_order),
                         std::move(new_null_precedence),
                         std::move(verticalized_col_depths));
}

/*
 * This helper function generates dremel data for any list-type columns in a
 * table. This data is necessary for lexicographic comparisons.
 */
auto list_lex_preprocess(table_view table, rmm::cuda_stream_view stream)
{
  std::vector<detail::dremel_data> dremel_data;
  std::vector<detail::dremel_device_view> dremel_device_views;
  for (auto const& col : table) {
    if (col.type().id() == type_id::LIST) {
      dremel_data.push_back(detail::get_dremel_data(col, {}, false, stream));
      dremel_device_views.push_back(dremel_data.back());
    }
  }
  auto d_dremel_device_views = detail::make_device_uvector_sync(dremel_device_views, stream);
  return std::make_tuple(std::move(dremel_data), std::move(d_dremel_device_views));
}

using column_checker_fn_t = std::function<void(column_view const&)>;

/**
 * @brief Check a table for compatibility with lexicographic comparison
 *
 * Checks whether a given table contains columns of non-relationally comparable types.
 */
void check_lex_compatibility(table_view const& input)
{
  // Basically check if there's any LIST of STRUCT or STRUCT of LIST hiding anywhere in the table
  column_checker_fn_t check_column = [&](column_view const& c) {
    if (c.type().id() == type_id::LIST) {
      auto const& list_col = lists_column_view(c);
      CUDF_EXPECTS(list_col.child().type().id() != type_id::STRUCT,
                   "Cannot lexicographic compare a table with a LIST of STRUCT column");
      check_column(list_col.child());
    } else if (c.type().id() == type_id::STRUCT) {
      for (auto child = c.child_begin(); child < c.child_end(); ++child) {
        CUDF_EXPECTS(child->type().id() != type_id::LIST,
                     "Cannot lexicographic compare a table with a STRUCT of LIST column");
        check_column(*child);
      }
    }
    if (not is_nested(c.type())) {
      CUDF_EXPECTS(is_relationally_comparable(c.type()),
                   "Cannot lexicographic compare a table with a column of type " +
                     jit::get_type_name(c.type()));
    }
  };
  for (column_view const& c : input) {
    check_column(c);
  }
}

/**
 * @brief Check a table for compatibility with equality comparison
 *
 * Checks whether a given table contains columns of non-equality comparable types.
 */
void check_eq_compatibility(table_view const& input)
{
  column_checker_fn_t check_column = [&](column_view const& c) {
    if (not is_nested(c.type())) {
      CUDF_EXPECTS(is_equality_comparable(c.type()),
                   "Cannot compare equality for a table with a column of type " +
                     jit::get_type_name(c.type()));
    }
    for (auto child = c.child_begin(); child < c.child_end(); ++child) {
      check_column(*child);
    }
  };
  for (column_view const& c : input) {
    check_column(c);
  }
}

void check_shape_compatibility(table_view const& lhs, table_view const& rhs)
{
  CUDF_EXPECTS(lhs.num_columns() == rhs.num_columns(),
               "Cannot compare tables with different number of columns");
  for (size_type i = 0; i < lhs.num_columns(); ++i) {
    CUDF_EXPECTS(column_types_equal(lhs.column(i), rhs.column(i)),
                 "Cannot compare tables with different column types");
  }
}

}  // namespace

namespace row {

namespace lexicographic {

std::shared_ptr<preprocessed_table> preprocessed_table::create(
  table_view const& t,
  host_span<order const> column_order,
  host_span<null_order const> null_precedence,
  rmm::cuda_stream_view stream)
{
  check_lex_compatibility(t);

  auto [verticalized_lhs, new_column_order, new_null_precedence, verticalized_col_depths] =
    decompose_structs(t, column_order, null_precedence);

  auto d_t               = table_device_view::create(verticalized_lhs, stream);
  auto d_column_order    = detail::make_device_uvector_async(new_column_order, stream);
  auto d_null_precedence = detail::make_device_uvector_async(new_null_precedence, stream);
  auto d_depths          = detail::make_device_uvector_async(verticalized_col_depths, stream);

  if (detail::has_nested_columns(t)) {
    auto [dremel_data, d_dremel_device_view] = list_lex_preprocess(verticalized_lhs, stream);
    return std::shared_ptr<preprocessed_table>(
      new preprocessed_table(std::move(d_t),
                             std::move(d_column_order),
                             std::move(d_null_precedence),
                             std::move(d_depths),
                             std::move(dremel_data),
                             std::move(d_dremel_device_view)));
  } else {
    return std::shared_ptr<preprocessed_table>(new preprocessed_table(std::move(d_t),
                                                                      std::move(d_column_order),
                                                                      std::move(d_null_precedence),
                                                                      std::move(d_depths)));
  }
}

two_table_comparator::two_table_comparator(table_view const& left,
                                           table_view const& right,
                                           host_span<order const> column_order,
                                           host_span<null_order const> null_precedence,
                                           rmm::cuda_stream_view stream)
  : d_left_table{preprocessed_table::create(left, column_order, null_precedence, stream)},
    d_right_table{preprocessed_table::create(right, column_order, null_precedence, stream)}
{
  check_shape_compatibility(left, right);
}

}  // namespace lexicographic

namespace equality {

std::shared_ptr<preprocessed_table> preprocessed_table::create(table_view const& t,
                                                               rmm::cuda_stream_view stream)
{
  check_eq_compatibility(t);

  auto [null_pushed_table, nullable_data] = structs::detail::push_down_nulls(t, stream);
  auto struct_offset_removed_table        = remove_struct_child_offsets(null_pushed_table);
  auto verticalized_t = std::get<0>(decompose_structs(struct_offset_removed_table));

  auto d_t = table_device_view_owner(table_device_view::create(verticalized_t, stream));
  return std::shared_ptr<preprocessed_table>(new preprocessed_table(
    std::move(d_t), std::move(nullable_data.new_null_masks), std::move(nullable_data.new_columns)));
}

two_table_comparator::two_table_comparator(table_view const& left,
                                           table_view const& right,
                                           rmm::cuda_stream_view stream)
  : d_left_table{preprocessed_table::create(left, stream)},
    d_right_table{preprocessed_table::create(right, stream)}
{
  check_shape_compatibility(left, right);
}

}  // namespace equality

}  // namespace row
}  // namespace experimental
}  // namespace cudf
