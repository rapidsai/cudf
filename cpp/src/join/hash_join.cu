#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2020-2021, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */
#include <thrust/iterator/discard_iterator.h>
#include <thrust/uninitialized_fill.h>
#include <join/hash_join.cuh>

#include <cudf/copying.hpp>
#include <cudf/detail/concatenate.cuh>
#include <cudf/detail/null_mask.hpp>
#include <cudf/detail/structs/utilities.hpp>

#include <rmm/cuda_stream_view.hpp>
#include <rmm/device_buffer.hpp>
#include <rmm/device_uvector.hpp>
#include <rmm/exec_policy.hpp>

#include <cstddef>
#include <iostream>
#include <numeric>

namespace cudf {
namespace detail {

namespace {

/**
 * @brief Device functor to determine if a row is valid.
 */
class row_is_valid {
 public:
  row_is_valid(bitmask_type const* row_bitmask) : _row_bitmask{row_bitmask} {}

  __device__ __inline__ bool operator()(const size_type& i) const noexcept
  {
    return cudf::bit_is_set(_row_bitmask, i);
  }

 private:
  bitmask_type const* _row_bitmask;
};

}  // anonymous namespace

std::pair<std::unique_ptr<table>, std::unique_ptr<table>> get_empty_joined_table(
  table_view const& probe, table_view const& build)
{
  std::unique_ptr<table> empty_probe = empty_like(probe);
  std::unique_ptr<table> empty_build = empty_like(build);
  return std::make_pair(std::move(empty_probe), std::move(empty_build));
}

/**
 * @brief Builds the hash table based on the given `build_table`.
 *
 * @param build Table of columns used to build join hash.
 * @param hash_table Build hash table.
 * @param compare_nulls Controls whether null join-key values should match or not.
 * @param stream CUDA stream used for device memory operations and kernel launches.
 *
 */
void build_join_hash_table(cudf::table_view const& build,
                           multimap_type& hash_table,
                           null_equality compare_nulls,
                           rmm::cuda_stream_view stream)
{
  auto build_table_ptr = cudf::table_device_view::create(build, stream);

  CUDF_EXPECTS(0 != build_table_ptr->num_columns(), "Selected build dataset is empty");
  CUDF_EXPECTS(0 != build_table_ptr->num_rows(), "Build side table has no rows");

  row_hash hash_build{*build_table_ptr};
  auto const empty_key_sentinel = hash_table.get_empty_key_sentinel();
  make_pair_function pair_func{hash_build, empty_key_sentinel};

  auto iter = cudf::detail::make_counting_transform_iterator(0, pair_func);

  size_type const build_table_num_rows{build_table_ptr->num_rows()};
  if ((compare_nulls == null_equality::EQUAL) or (not nullable(build))) {
    hash_table.insert(iter, iter + build_table_num_rows, stream.value());
  } else {
    thrust::counting_iterator<size_type> stencil(0);
    auto const row_bitmask = cudf::detail::bitmask_and(build, stream).first;
    row_is_valid pred{static_cast<bitmask_type const*>(row_bitmask.data())};

    // insert valid rows
    hash_table.insert_if(iter, iter + build_table_num_rows, stencil, pred, stream.value());
  }
}

/**
 * @brief Probes the `hash_table` built from `build_table` for tuples in `probe_table`,
 * and returns the output indices of `build_table` and `probe_table` as a combined table.
 * Behavior is undefined if the provided `output_size` is smaller than the actual output size.
 *
 * @tparam JoinKind The type of join to be performed.
 *
 * @param build_table Table of build side columns to join.
 * @param probe_table Table of probe side columns to join.
 * @param hash_table Hash table built from `build_table`.
 * @param compare_nulls Controls whether null join-key values should match or not.
 * @param output_size Optional value which allows users to specify the exact output size.
 * @param stream CUDA stream used for device memory operations and kernel launches.
 * @param mr Device memory resource used to allocate the returned vectors.
 *
 * @return Join output indices vector pair.
 */
template <join_kind JoinKind>
std::pair<std::unique_ptr<rmm::device_uvector<size_type>>,
          std::unique_ptr<rmm::device_uvector<size_type>>>
probe_join_hash_table(cudf::table_device_view build_table,
                      cudf::table_device_view probe_table,
                      multimap_type const& hash_table,
                      null_equality compare_nulls,
                      std::optional<std::size_t> output_size,
                      rmm::cuda_stream_view stream,
                      rmm::mr::device_memory_resource* mr)
{
  // Use the output size directly if provided. Otherwise, compute the exact output size
  constexpr cudf::detail::join_kind ProbeJoinKind = (JoinKind == cudf::detail::join_kind::FULL_JOIN)
                                                      ? cudf::detail::join_kind::LEFT_JOIN
                                                      : JoinKind;

  std::size_t const join_size = output_size
                                  ? *output_size
                                  : compute_join_output_size<ProbeJoinKind>(
                                      build_table, probe_table, hash_table, compare_nulls, stream);

  // If output size is zero, return immediately
  if (join_size == 0) {
    return std::make_pair(std::make_unique<rmm::device_uvector<size_type>>(0, stream, mr),
                          std::make_unique<rmm::device_uvector<size_type>>(0, stream, mr));
  }

  auto left_indices  = std::make_unique<rmm::device_uvector<size_type>>(join_size, stream, mr);
  auto right_indices = std::make_unique<rmm::device_uvector<size_type>>(join_size, stream, mr);

  pair_equality equality{probe_table, build_table, compare_nulls == null_equality::EQUAL};

  row_hash hash_probe{probe_table};
  auto const empty_key_sentinel = hash_table.get_empty_key_sentinel();
  make_pair_function pair_func{hash_probe, empty_key_sentinel};

  auto iter = cudf::detail::make_counting_transform_iterator(0, pair_func);

  const cudf::size_type probe_table_num_rows = probe_table.num_rows();

  auto out1_zip_begin = thrust::make_zip_iterator(
    thrust::make_tuple(thrust::make_discard_iterator(), left_indices->begin()));
  auto out2_zip_begin = thrust::make_zip_iterator(
    thrust::make_tuple(thrust::make_discard_iterator(), right_indices->begin()));

  if constexpr (JoinKind == cudf::detail::join_kind::FULL_JOIN or
                JoinKind == cudf::detail::join_kind::LEFT_JOIN) {
    [[maybe_unused]] auto [out1_zip_end, out2_zip_end] = hash_table.pair_retrieve_outer(
      iter, iter + probe_table_num_rows, out1_zip_begin, out2_zip_begin, equality, stream.value());

    if constexpr (JoinKind == cudf::detail::join_kind::FULL_JOIN) {
      auto const actual_size = out1_zip_end - out1_zip_begin;
      left_indices->resize(actual_size, stream);
      right_indices->resize(actual_size, stream);
    }
  } else {
    hash_table.pair_retrieve(
      iter, iter + probe_table_num_rows, out1_zip_begin, out2_zip_begin, equality, stream.value());
  }
  return std::make_pair(std::move(left_indices), std::move(right_indices));
}

/**
 * @brief Probes the `hash_table` built from `build_table` for tuples in `probe_table` twice,
 * and returns the output size of a full join operation between `build_table` and `probe_table`.
 * TODO: this is a temporary solution as part of `full_join_size`. To be refactored during
 * cuco integration.
 *
 * @param build_table Table of build side columns to join.
 * @param probe_table Table of probe side columns to join.
 * @param hash_table Hash table built from `build_table`.
 * @param compare_nulls Controls whether null join-key values should match or not.
 * @param stream CUDA stream used for device memory operations and kernel launches.
 * @param mr Device memory resource used to allocate the intermediate vectors.
 *
 * @return Output size of full join.
 */
std::size_t get_full_join_size(cudf::table_device_view build_table,
                               cudf::table_device_view probe_table,
                               multimap_type const& hash_table,
                               null_equality compare_nulls,
                               rmm::cuda_stream_view stream,
                               rmm::mr::device_memory_resource* mr)
{
  std::size_t join_size = compute_join_output_size<cudf::detail::join_kind::LEFT_JOIN>(
    build_table, probe_table, hash_table, compare_nulls, stream);

  // If output size is zero, return immediately
  if (join_size == 0) { return join_size; }

  rmm::device_scalar<size_type> write_index(0, stream);

  auto left_indices  = std::make_unique<rmm::device_uvector<size_type>>(join_size, stream, mr);
  auto right_indices = std::make_unique<rmm::device_uvector<size_type>>(join_size, stream, mr);

  pair_equality equality{probe_table, build_table, compare_nulls == null_equality::EQUAL};

  row_hash hash_probe{probe_table};
  auto const empty_key_sentinel = hash_table.get_empty_key_sentinel();
  make_pair_function pair_func{hash_probe, empty_key_sentinel};

  auto iter = cudf::detail::make_counting_transform_iterator(0, pair_func);

  const cudf::size_type probe_table_num_rows = probe_table.num_rows();

  auto out1_zip_begin = thrust::make_zip_iterator(
    thrust::make_tuple(thrust::make_discard_iterator(), left_indices->begin()));
  auto out2_zip_begin = thrust::make_zip_iterator(
    thrust::make_tuple(thrust::make_discard_iterator(), right_indices->begin()));

  hash_table.pair_retrieve_outer(
    iter, iter + probe_table_num_rows, out1_zip_begin, out2_zip_begin, equality, stream.value());

  // Release intermediate memory allocation
  left_indices->resize(0, stream);

  auto const left_table_row_count  = probe_table.num_rows();
  auto const right_table_row_count = build_table.num_rows();

  std::size_t left_join_complement_size;

  // If left table is empty then all rows of the right table should be represented in the joined
  // indices.
  if (left_table_row_count == 0) {
    left_join_complement_size = right_table_row_count;
  } else {
    // Assume all the indices in invalid_index_map are invalid
    auto invalid_index_map =
      std::make_unique<rmm::device_uvector<size_type>>(right_table_row_count, stream);
    thrust::uninitialized_fill(
      rmm::exec_policy(stream), invalid_index_map->begin(), invalid_index_map->end(), int32_t{1});

    // Functor to check for index validity since left joins can create invalid indices
    valid_range<size_type> valid(0, right_table_row_count);

    // invalid_index_map[index_ptr[i]] = 0 for i = 0 to right_table_row_count
    // Thus specifying that those locations are valid
    thrust::scatter_if(rmm::exec_policy(stream),
                       thrust::make_constant_iterator(0),
                       thrust::make_constant_iterator(0) + right_indices->size(),
                       right_indices->begin(),      // Index locations
                       right_indices->begin(),      // Stencil - Check if index location is valid
                       invalid_index_map->begin(),  // Output indices
                       valid);                      // Stencil Predicate

    // Create list of indices that have been marked as invalid
    left_join_complement_size = thrust::count_if(rmm::exec_policy(stream),
                                                 invalid_index_map->begin(),
                                                 invalid_index_map->end(),
                                                 thrust::identity<size_type>());
  }
  return join_size + left_join_complement_size;
}

std::unique_ptr<cudf::table> combine_table_pair(std::unique_ptr<cudf::table>&& left,
                                                std::unique_ptr<cudf::table>&& right)
{
  auto joined_cols = left->release();
  auto right_cols  = right->release();
  joined_cols.insert(joined_cols.end(),
                     std::make_move_iterator(right_cols.begin()),
                     std::make_move_iterator(right_cols.end()));
  return std::make_unique<cudf::table>(std::move(joined_cols));
}

}  // namespace detail

hash_join::hash_join_impl::~hash_join_impl() = default;

hash_join::hash_join_impl::hash_join_impl(cudf::table_view const& build,
                                          null_equality compare_nulls,
                                          rmm::cuda_stream_view stream)
  : _is_empty{build.num_rows() == 0},
    _hash_table{compute_hash_table_size(build.num_rows()),
                std::numeric_limits<hash_value_type>::max(),
                cudf::detail::JoinNoneValue,
                stream.value()}
{
  CUDF_FUNC_RANGE();
  CUDF_EXPECTS(0 != build.num_columns(), "Hash join build table is empty");
  CUDF_EXPECTS(build.num_rows() < cudf::detail::MAX_JOIN_SIZE,
               "Build column size is too big for hash join");

  // need to store off the owning structures for some of the views in _build
  _flattened_build_table = structs::detail::flatten_nested_columns(
    build, {}, {}, structs::detail::column_nullability::FORCE);
  _build = _flattened_build_table;

  if (_is_empty) { return; }

  build_join_hash_table(_build, _hash_table, compare_nulls, stream);
}

std::pair<std::unique_ptr<rmm::device_uvector<size_type>>,
          std::unique_ptr<rmm::device_uvector<size_type>>>
hash_join::hash_join_impl::inner_join(cudf::table_view const& probe,
                                      null_equality compare_nulls,
                                      std::optional<std::size_t> output_size,
                                      rmm::cuda_stream_view stream,
                                      rmm::mr::device_memory_resource* mr) const
{
  CUDF_FUNC_RANGE();
  return compute_hash_join<cudf::detail::join_kind::INNER_JOIN>(
    probe, compare_nulls, output_size, stream, mr);
}

std::pair<std::unique_ptr<rmm::device_uvector<size_type>>,
          std::unique_ptr<rmm::device_uvector<size_type>>>
hash_join::hash_join_impl::left_join(cudf::table_view const& probe,
                                     null_equality compare_nulls,
                                     std::optional<std::size_t> output_size,
                                     rmm::cuda_stream_view stream,
                                     rmm::mr::device_memory_resource* mr) const
{
  CUDF_FUNC_RANGE();
  return compute_hash_join<cudf::detail::join_kind::LEFT_JOIN>(
    probe, compare_nulls, output_size, stream, mr);
}

std::pair<std::unique_ptr<rmm::device_uvector<size_type>>,
          std::unique_ptr<rmm::device_uvector<size_type>>>
hash_join::hash_join_impl::full_join(cudf::table_view const& probe,
                                     null_equality compare_nulls,
                                     std::optional<std::size_t> output_size,
                                     rmm::cuda_stream_view stream,
                                     rmm::mr::device_memory_resource* mr) const
{
  CUDF_FUNC_RANGE();
  return compute_hash_join<cudf::detail::join_kind::FULL_JOIN>(
    probe, compare_nulls, output_size, stream, mr);
}

std::size_t hash_join::hash_join_impl::inner_join_size(cudf::table_view const& probe,
                                                       null_equality compare_nulls,
                                                       rmm::cuda_stream_view stream) const
{
  CUDF_FUNC_RANGE();

  // Return directly if build table is empty
  if (_is_empty) { return 0; }

  auto flattened_probe = structs::detail::flatten_nested_columns(
    probe, {}, {}, structs::detail::column_nullability::FORCE);
  auto const flattened_probe_table = flattened_probe.flattened_columns();

  auto build_table_ptr           = cudf::table_device_view::create(_build, stream);
  auto flattened_probe_table_ptr = cudf::table_device_view::create(flattened_probe_table, stream);

  return cudf::detail::compute_join_output_size<cudf::detail::join_kind::INNER_JOIN>(
    *build_table_ptr, *flattened_probe_table_ptr, _hash_table, compare_nulls, stream);
}

std::size_t hash_join::hash_join_impl::left_join_size(cudf::table_view const& probe,
                                                      null_equality compare_nulls,
                                                      rmm::cuda_stream_view stream) const
{
  CUDF_FUNC_RANGE();

  // Trivial left join case - exit early
  if (_is_empty) { return probe.num_rows(); }

  auto flattened_probe = structs::detail::flatten_nested_columns(
    probe, {}, {}, structs::detail::column_nullability::FORCE);
  auto const flattened_probe_table = flattened_probe.flattened_columns();

  auto build_table_ptr           = cudf::table_device_view::create(_build, stream);
  auto flattened_probe_table_ptr = cudf::table_device_view::create(flattened_probe_table, stream);

  return cudf::detail::compute_join_output_size<cudf::detail::join_kind::LEFT_JOIN>(
    *build_table_ptr, *flattened_probe_table_ptr, _hash_table, compare_nulls, stream);
}

std::size_t hash_join::hash_join_impl::full_join_size(cudf::table_view const& probe,
                                                      null_equality compare_nulls,
                                                      rmm::cuda_stream_view stream,
                                                      rmm::mr::device_memory_resource* mr) const
{
  CUDF_FUNC_RANGE();

  // Trivial left join case - exit early
  if (_is_empty) { return probe.num_rows(); }

  auto flattened_probe = structs::detail::flatten_nested_columns(
    probe, {}, {}, structs::detail::column_nullability::FORCE);
  auto const flattened_probe_table = flattened_probe.flattened_columns();

  auto build_table_ptr           = cudf::table_device_view::create(_build, stream);
  auto flattened_probe_table_ptr = cudf::table_device_view::create(flattened_probe_table, stream);

  return get_full_join_size(
    *build_table_ptr, *flattened_probe_table_ptr, _hash_table, compare_nulls, stream, mr);
}

template <cudf::detail::join_kind JoinKind>
std::pair<std::unique_ptr<rmm::device_uvector<size_type>>,
          std::unique_ptr<rmm::device_uvector<size_type>>>
hash_join::hash_join_impl::compute_hash_join(cudf::table_view const& probe,
                                             null_equality compare_nulls,
                                             std::optional<std::size_t> output_size,
                                             rmm::cuda_stream_view stream,
                                             rmm::mr::device_memory_resource* mr) const
{
  CUDF_EXPECTS(0 != probe.num_columns(), "Hash join probe table is empty");
  CUDF_EXPECTS(probe.num_rows() < cudf::detail::MAX_JOIN_SIZE,
               "Probe column size is too big for hash join");

  auto flattened_probe = structs::detail::flatten_nested_columns(
    probe, {}, {}, structs::detail::column_nullability::FORCE);
  auto const flattened_probe_table = flattened_probe.flattened_columns();

  CUDF_EXPECTS(_build.num_columns() == flattened_probe_table.num_columns(),
               "Mismatch in number of columns to be joined on");

  if (is_trivial_join(flattened_probe_table, _build, JoinKind)) {
    return std::make_pair(std::make_unique<rmm::device_uvector<size_type>>(0, stream, mr),
                          std::make_unique<rmm::device_uvector<size_type>>(0, stream, mr));
  }

  CUDF_EXPECTS(std::equal(std::cbegin(_build),
                          std::cend(_build),
                          std::cbegin(flattened_probe_table),
                          std::cend(flattened_probe_table),
                          [](const auto& b, const auto& p) { return b.type() == p.type(); }),
               "Mismatch in joining column data types");

  return probe_join_indices<JoinKind>(
    flattened_probe_table, compare_nulls, output_size, stream, mr);
}

template <cudf::detail::join_kind JoinKind>
std::pair<std::unique_ptr<rmm::device_uvector<size_type>>,
          std::unique_ptr<rmm::device_uvector<size_type>>>
hash_join::hash_join_impl::probe_join_indices(cudf::table_view const& probe,
                                              null_equality compare_nulls,
                                              std::optional<std::size_t> output_size,
                                              rmm::cuda_stream_view stream,
                                              rmm::mr::device_memory_resource* mr) const
{
  // Trivial left join case - exit early
  if (_is_empty and JoinKind != cudf::detail::join_kind::INNER_JOIN) {
    return get_trivial_left_join_indices(probe, stream, mr);
  }

  CUDF_EXPECTS(!_is_empty, "Hash table of hash join is null.");

  auto build_table_ptr = cudf::table_device_view::create(_build, stream);
  auto probe_table_ptr = cudf::table_device_view::create(probe, stream);

  auto join_indices = cudf::detail::probe_join_hash_table<JoinKind>(
    *build_table_ptr, *probe_table_ptr, _hash_table, compare_nulls, output_size, stream, mr);

  if constexpr (JoinKind == cudf::detail::join_kind::FULL_JOIN) {
    auto complement_indices = detail::get_left_join_indices_complement(
      join_indices.second, probe.num_rows(), _build.num_rows(), stream, mr);
    join_indices = detail::concatenate_vector_pairs(join_indices, complement_indices, stream);
  }
  return join_indices;
}

}  // namespace cudf
