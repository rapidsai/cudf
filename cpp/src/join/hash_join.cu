#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2020-2025, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */
#include "join_common_utils.cuh"

#include <cudf/copying.hpp>
#include <cudf/detail/iterator.cuh>
#include <cudf/detail/join/hash_join.cuh>
#include <cudf/detail/nvtx/ranges.hpp>
#include <cudf/detail/structs/utilities.hpp>
#include <cudf/hashing/detail/helper_functions.cuh>
#include <cudf/join/hash_join.hpp>
#include <cudf/table/experimental/row_operators.cuh>
#include <cudf/table/primitive_row_operators.cuh>
#include <cudf/utilities/error.hpp>
#include <cudf/utilities/memory_resource.hpp>
#include <cudf/utilities/type_checks.hpp>

#include <rmm/cuda_stream_view.hpp>
#include <rmm/device_buffer.hpp>
#include <rmm/device_uvector.hpp>
#include <rmm/exec_policy.hpp>

#include <cuda/std/functional>
#include <cuda/std/iterator>
#include <thrust/count.h>
#include <thrust/iterator/constant_iterator.h>
#include <thrust/iterator/discard_iterator.h>
#include <thrust/iterator/transform_output_iterator.h>
#include <thrust/scatter.h>
#include <thrust/tuple.h>
#include <thrust/uninitialized_fill.h>

#include <cstddef>
#include <memory>

namespace cudf {
namespace detail {
namespace {
using hash_table_t = cudf::hash_join::impl_type::hash_table_t;

// Multimap type used for mixed joins. TODO: This is a temporary alias used
// TODO: `pair_equal` and `pair_fn` to be moved to common utils during
// mixed-join migration
template <typename Hasher>
struct pair_fn {
  pair_fn(Hasher hash) : _hash{std::move(hash)} {}

  __device__ cuco::pair<hash_value_type, size_type> operator()(size_type i) const noexcept
  {
    return cuco::pair{_hash(i), i};
  }

 private:
  Hasher _hash;
};

template <typename Equal>
class pair_equal {
 public:
  pair_equal(Equal check_row_equality) : _check_row_equality{std::move(check_row_equality)} {}

  __device__ __forceinline__ bool operator()(
    cuco::pair<hash_value_type, size_type> const& lhs,
    cuco::pair<hash_value_type, size_type> const& rhs) const noexcept
  {
    using experimental::row::lhs_index_type;
    using experimental::row::rhs_index_type;

    return lhs.first == rhs.first and
           _check_row_equality(lhs_index_type{lhs.second}, rhs_index_type{rhs.second});
  }

 private:
  Equal _check_row_equality;
};

struct output_fn {
  __device__ constexpr cudf::size_type operator()(
    cuco::pair<hash_value_type, cudf::size_type> const& slot) const
  {
    return slot.second;
  }
};

class primitive_pair_equal {
 public:
  primitive_pair_equal(cudf::row::primitive::row_equality_comparator check_row_equality)
    : _check_row_equality{std::move(check_row_equality)}
  {
  }

  __device__ __forceinline__ bool operator()(
    cuco::pair<hash_value_type, size_type> const& lhs,
    cuco::pair<hash_value_type, size_type> const& rhs) const noexcept
  {
    return lhs.first == rhs.first and _check_row_equality(lhs.second, rhs.second);
  }

 private:
  cudf::row::primitive::row_equality_comparator _check_row_equality;
};

/**
 * @brief Builds a hash table from the input build table for performing hash joins
 *
 * @throw std::invalid_argument if build table is empty or has no columns
 *
 * @param build The build-side table containing columns to hash and join on
 * @param preprocessed_build Pre-processed version of build table optimized for row operations
 * @param hash_table The hash table to populate with build table rows
 * @param has_nested_nulls Whether the build table contains any nested null values
 * @param nulls_equal How to handle null values during join - EQUAL means nulls match other nulls
 * @param bitmask Validity bitmask indicating which build table rows are valid/non-null
 * @param stream CUDA stream to use for device operations
 */
void build_hash_join(
  cudf::table_view const& build,
  std::shared_ptr<experimental::row::equality::preprocessed_table> const& preprocessed_build,
  cudf::detail::hash_table_t& hash_table,
  bool has_nested_nulls,
  null_equality nulls_equal,
  [[maybe_unused]] bitmask_type const* bitmask,
  rmm::cuda_stream_view stream)
{
  CUDF_EXPECTS(0 != build.num_columns(), "Selected build dataset is empty", std::invalid_argument);
  CUDF_EXPECTS(0 != build.num_rows(), "Build side table has no rows", std::invalid_argument);

  // Lambda to insert rows into hash table
  auto insert_rows = [&](auto const& build, auto const& d_hasher) {
    auto const iter = cudf::detail::make_counting_transform_iterator(0, pair_fn{d_hasher});

    if (nulls_equal == cudf::null_equality::EQUAL or not nullable(build)) {
      hash_table.insert(iter, iter + build.num_rows(), stream.value());
    } else {
      auto const stencil = thrust::counting_iterator<size_type>{0};
      auto const pred    = row_is_valid{bitmask};

      // insert valid rows
      hash_table.insert_if(iter, iter + build.num_rows(), stencil, pred, stream.value());
    }
  };

  auto const nulls = nullate::DYNAMIC{has_nested_nulls};

  // Insert rows into hash table
  if (cudf::is_primitive_row_op_compatible(build)) {
    auto const d_hasher = cudf::row::primitive::row_hasher{nulls, preprocessed_build};

    insert_rows(build, d_hasher);
  } else {
    auto const row_hash = experimental::row::hash::row_hasher{preprocessed_build};
    auto const d_hasher = row_hash.device_hasher(nulls);

    insert_rows(build, d_hasher);
  }
}

/**
 * @brief Calculates the exact size of the join output produced when
 * joining two tables together.
 *
 * @throw cudf::logic_error if join is not INNER_JOIN or LEFT_JOIN
 *
 * @param build_table The right hand table
 * @param probe_table The left hand table
 * @param preprocessed_build shared_ptr to cudf::experimental::row::equality::preprocessed_table for
 *                           build_table
 * @param preprocessed_probe shared_ptr to cudf::experimental::row::equality::preprocessed_table for
 *                           probe_table
 * @param hash_table A hash table built on the build table that maps the index
 *                   of every row to the hash value of that row
 * @param join The type of join to be performed
 * @param has_nulls Flag to denote if build or probe tables have nested nulls
 * @param nulls_equal Flag to denote nulls are equal or not
 * @param stream CUDA stream used for device memory operations and kernel launches
 *
 * @return The exact size of the output of the join operation
 */
std::size_t compute_join_output_size(
  table_view const& build_table,
  table_view const& probe_table,
  std::shared_ptr<cudf::experimental::row::equality::preprocessed_table> const& preprocessed_build,
  std::shared_ptr<cudf::experimental::row::equality::preprocessed_table> const& preprocessed_probe,
  cudf::detail::hash_table_t const& hash_table,
  join_kind join,
  bool has_nulls,
  cudf::null_equality nulls_equal,
  rmm::cuda_stream_view stream)
{
  size_type const build_table_num_rows{build_table.num_rows()};
  size_type const probe_table_num_rows{probe_table.num_rows()};

  // If the build table is empty, we know exactly how large the output
  // will be for the different types of joins and can return immediately
  if (0 == build_table_num_rows) {
    switch (join) {
      // Inner join with an empty table will have no output
      case join_kind::INNER_JOIN: return 0;

      // Left join with an empty table will have an output of NULL rows
      // equal to the number of rows in the probe table
      case join_kind::LEFT_JOIN: return probe_table_num_rows;

      default: CUDF_FAIL("Unsupported join type");
    }
  }

  auto const probe_nulls = cudf::nullate::DYNAMIC{has_nulls};

  // Common function to handle both primitive and non-primitive cases
  auto compute_size = [&](auto equality, auto d_hasher) {
    auto const iter = cudf::detail::make_counting_transform_iterator(0, pair_fn{d_hasher});

    if (join == join_kind::LEFT_JOIN) {
      return hash_table.count_outer(
        iter, iter + probe_table_num_rows, equality, hash_table.hash_function(), stream.value());
    } else {
      return hash_table.count(
        iter, iter + probe_table_num_rows, equality, hash_table.hash_function(), stream.value());
    }
  };

  // Use primitive row operator logic if build table is compatible. Otherwise, use non-primitive row
  // operator logic.
  if (cudf::is_primitive_row_op_compatible(build_table)) {
    auto const d_hasher = cudf::row::primitive::row_hasher{probe_nulls, preprocessed_probe};
    auto const d_equal  = cudf::row::primitive::row_equality_comparator{
      probe_nulls, preprocessed_probe, preprocessed_build, nulls_equal};

    return compute_size(primitive_pair_equal{d_equal}, d_hasher);
  } else {
    auto const d_hasher =
      cudf::experimental::row::hash::row_hasher{preprocessed_probe}.device_hasher(probe_nulls);
    auto const row_comparator = cudf::experimental::row::equality::two_table_comparator{
      preprocessed_probe, preprocessed_build};

    if (cudf::detail::has_nested_columns(probe_table)) {
      auto const d_equal = row_comparator.equal_to<true>(has_nulls, nulls_equal);
      return compute_size(pair_equal{d_equal}, d_hasher);
    } else {
      auto const d_equal = row_comparator.equal_to<false>(has_nulls, nulls_equal);
      return compute_size(pair_equal{d_equal}, d_hasher);
    }
  }
}

/**
 * @brief Probes the `hash_table` built from `build_table` for tuples in `probe_table`,
 * and returns the output indices of `build_table` and `probe_table` as a combined table.
 * Behavior is undefined if the provided `output_size` is smaller than the actual output size.
 *
 * @param build_table Table of build side columns to join
 * @param probe_table Table of probe side columns to join
 * @param preprocessed_build shared_ptr to cudf::experimental::row::equality::preprocessed_table
 * for build_table
 * @param preprocessed_probe shared_ptr to cudf::experimental::row::equality::preprocessed_table
 * for probe_table
 * @param hash_table Hash table built from `build_table`
 * @param join The type of join to be performed
 * @param has_nulls Flag to denote if build or probe tables have nested nulls
 * @param compare_nulls Controls whether null join-key values should match or not
 * @param output_size Optional value which allows users to specify the exact output size
 * @param stream CUDA stream used for device memory operations and kernel launches
 * @param mr Device memory resource used to allocate the returned vectors
 *
 * @return Join output indices vector pair.
 */
std::pair<std::unique_ptr<rmm::device_uvector<size_type>>,
          std::unique_ptr<rmm::device_uvector<size_type>>>
probe_join_hash_table(
  cudf::table_view const& build_table,
  cudf::table_view const& probe_table,
  std::shared_ptr<cudf::experimental::row::equality::preprocessed_table> const& preprocessed_build,
  std::shared_ptr<cudf::experimental::row::equality::preprocessed_table> const& preprocessed_probe,
  cudf::detail::hash_table_t const& hash_table,
  join_kind join,
  bool has_nulls,
  null_equality compare_nulls,
  std::optional<std::size_t> output_size,
  rmm::cuda_stream_view stream,
  rmm::device_async_resource_ref mr)
{
  // Use the output size directly if provided. Otherwise, compute the exact output size
  auto const probe_join_type =
    (join == cudf::detail::join_kind::FULL_JOIN) ? cudf::detail::join_kind::LEFT_JOIN : join;

  std::size_t const join_size = output_size ? *output_size
                                            : compute_join_output_size(build_table,
                                                                       probe_table,
                                                                       preprocessed_build,
                                                                       preprocessed_probe,
                                                                       hash_table,
                                                                       probe_join_type,
                                                                       has_nulls,
                                                                       compare_nulls,
                                                                       stream);

  // If output size is zero, return immediately
  if (join_size == 0) {
    return std::pair(std::make_unique<rmm::device_uvector<size_type>>(0, stream, mr),
                     std::make_unique<rmm::device_uvector<size_type>>(0, stream, mr));
  }

  auto left_indices  = std::make_unique<rmm::device_uvector<size_type>>(join_size, stream, mr);
  auto right_indices = std::make_unique<rmm::device_uvector<size_type>>(join_size, stream, mr);
  cudf::experimental::prefetch::detail::prefetch("hash_join", *left_indices, stream);
  cudf::experimental::prefetch::detail::prefetch("hash_join", *right_indices, stream);

  auto const probe_table_num_rows = probe_table.num_rows();
  auto const out_probe_begin =
    thrust::make_transform_output_iterator(left_indices->begin(), output_fn{});
  auto const out_build_begin =
    thrust::make_transform_output_iterator(right_indices->begin(), output_fn{});

  // Common function to handle retrieval for both primitive and non-primitive cases
  auto retrieve_results = [&](auto equality, auto iter) {
    if (join == cudf::detail::join_kind::FULL_JOIN || join == cudf::detail::join_kind::LEFT_JOIN) {
      [[maybe_unused]] auto out_probe_end = hash_table
                                              .retrieve_outer(iter,
                                                              iter + probe_table_num_rows,
                                                              equality,
                                                              hash_table.hash_function(),
                                                              out_probe_begin,
                                                              out_build_begin,
                                                              stream.value())
                                              .first;

      if (join == cudf::detail::join_kind::FULL_JOIN) {
        auto const actual_size = cuda::std::distance(out_probe_begin, out_probe_end);
        left_indices->resize(actual_size, stream);
        right_indices->resize(actual_size, stream);
      }
    } else {
      hash_table.retrieve(iter,
                          iter + probe_table_num_rows,
                          equality,
                          hash_table.hash_function(),
                          out_probe_begin,
                          out_build_begin,
                          stream.value());
    }
  };

  auto const probe_nulls = cudf::nullate::DYNAMIC{has_nulls};

  if (cudf::is_primitive_row_op_compatible(build_table)) {
    auto const d_hasher = cudf::row::primitive::row_hasher{probe_nulls, preprocessed_probe};
    auto const d_equal  = cudf::row::primitive::row_equality_comparator{
      probe_nulls, preprocessed_probe, preprocessed_build, compare_nulls};
    auto const iter = cudf::detail::make_counting_transform_iterator(0, pair_fn{d_hasher});

    retrieve_results(primitive_pair_equal{d_equal}, iter);
  } else {
    auto const d_hasher =
      cudf::experimental::row::hash::row_hasher{preprocessed_probe}.device_hasher(probe_nulls);
    auto const iter = cudf::detail::make_counting_transform_iterator(0, pair_fn{d_hasher});

    auto const row_comparator = cudf::experimental::row::equality::two_table_comparator{
      preprocessed_probe, preprocessed_build};

    if (cudf::detail::has_nested_columns(probe_table)) {
      auto const d_equal = row_comparator.equal_to<true>(probe_nulls, compare_nulls);
      retrieve_results(pair_equal{d_equal}, iter);
    } else {
      auto const d_equal = row_comparator.equal_to<false>(probe_nulls, compare_nulls);
      retrieve_results(pair_equal{d_equal}, iter);
    }
  }

  return std::pair(std::move(left_indices), std::move(right_indices));
}

/**
 * @brief Probes the `hash_table` built from `build_table` for tuples in `probe_table` twice,
 * and returns the output size of a full join operation between `build_table` and `probe_table`.
 * TODO: this is a temporary solution as part of `full_join_size`. To be refactored during
 * cuco integration.
 *
 * @param build_table Table of build side columns to join
 * @param probe_table Table of probe side columns to join
 * @param preprocessed_build shared_ptr to cudf::experimental::row::equality::preprocessed_table
 * for build_table
 * @param preprocessed_probe shared_ptr to cudf::experimental::row::equality::preprocessed_table
 * for probe_table
 * @param hash_table Hash table built from `build_table`
 * @param has_nulls Flag to denote if build or probe tables have nested nulls
 * @param compare_nulls Controls whether null join-key values should match or not
 * @param stream CUDA stream used for device memory operations and kernel launches
 * @param mr Device memory resource used to allocate the intermediate vectors
 *
 * @return Output size of full join.
 */
std::size_t get_full_join_size(
  cudf::table_view const& build_table,
  cudf::table_view const& probe_table,
  std::shared_ptr<cudf::experimental::row::equality::preprocessed_table> const& preprocessed_build,
  std::shared_ptr<cudf::experimental::row::equality::preprocessed_table> const& preprocessed_probe,
  cudf::detail::hash_table_t const& hash_table,
  bool has_nulls,
  null_equality compare_nulls,
  rmm::cuda_stream_view stream,
  rmm::device_async_resource_ref mr)
{
  std::size_t join_size = compute_join_output_size(build_table,
                                                   probe_table,
                                                   preprocessed_build,
                                                   preprocessed_probe,
                                                   hash_table,
                                                   cudf::detail::join_kind::LEFT_JOIN,
                                                   has_nulls,
                                                   compare_nulls,
                                                   stream);

  // If output size is zero, return immediately
  if (join_size == 0) { return join_size; }

  auto right_indices = std::make_unique<rmm::device_uvector<size_type>>(join_size, stream, mr);

  auto const probe_nulls = cudf::nullate::DYNAMIC{has_nulls};

  cudf::size_type const probe_table_num_rows = probe_table.num_rows();

  auto const out_build_begin =
    thrust::make_transform_output_iterator(right_indices->begin(), output_fn{});

  // Apply primitive row operator logic
  if (cudf::is_primitive_row_op_compatible(build_table)) {
    auto const d_hasher = cudf::row::primitive::row_hasher{probe_nulls, preprocessed_probe};
    auto const d_equal  = cudf::row::primitive::row_equality_comparator{
      probe_nulls, preprocessed_probe, preprocessed_build, compare_nulls};
    auto const iter     = cudf::detail::make_counting_transform_iterator(0, pair_fn{d_hasher});
    auto const equality = primitive_pair_equal{d_equal};

    hash_table.retrieve_outer(iter,
                              iter + probe_table_num_rows,
                              equality,
                              hash_table.hash_function(),
                              thrust::make_discard_iterator(),
                              out_build_begin,
                              stream.value());
  } else {
    auto const d_hasher =
      cudf::experimental::row::hash::row_hasher{preprocessed_probe}.device_hasher(probe_nulls);
    auto const iter = cudf::detail::make_counting_transform_iterator(0, pair_fn{d_hasher});

    auto const row_comparator = cudf::experimental::row::equality::two_table_comparator{
      preprocessed_probe, preprocessed_build};
    auto const comparator_helper = [&](auto d_equal) {
      auto const equality = pair_equal{d_equal};
      hash_table.retrieve_outer(iter,
                                iter + probe_table_num_rows,
                                equality,
                                hash_table.hash_function(),
                                thrust::make_discard_iterator(),
                                out_build_begin,
                                stream.value());
    };
    if (cudf::detail::has_nested_columns(probe_table)) {
      auto const d_equal = row_comparator.equal_to<true>(probe_nulls, compare_nulls);
      comparator_helper(d_equal);
    } else {
      auto const d_equal = row_comparator.equal_to<false>(probe_nulls, compare_nulls);
      comparator_helper(d_equal);
    }
  }

  auto const left_table_row_count  = probe_table.num_rows();
  auto const right_table_row_count = build_table.num_rows();

  std::size_t left_join_complement_size;

  // If left table is empty then all rows of the right table should be represented in the joined
  // indices.
  if (left_table_row_count == 0) {
    left_join_complement_size = right_table_row_count;
  } else {
    // Assume all the indices in invalid_index_map are invalid
    auto invalid_index_map =
      std::make_unique<rmm::device_uvector<size_type>>(right_table_row_count, stream);
    thrust::uninitialized_fill(rmm::exec_policy_nosync(stream),
                               invalid_index_map->begin(),
                               invalid_index_map->end(),
                               int32_t{1});

    // Functor to check for index validity since left joins can create invalid indices
    valid_range<size_type> valid(0, right_table_row_count);

    // invalid_index_map[index_ptr[i]] = 0 for i = 0 to right_table_row_count
    // Thus specifying that those locations are valid
    thrust::scatter_if(rmm::exec_policy_nosync(stream),
                       thrust::make_constant_iterator(0),
                       thrust::make_constant_iterator(0) + right_indices->size(),
                       right_indices->begin(),      // Index locations
                       right_indices->begin(),      // Stencil - Check if index location is valid
                       invalid_index_map->begin(),  // Output indices
                       valid);                      // Stencil Predicate

    // Create list of indices that have been marked as invalid
    left_join_complement_size = thrust::count_if(rmm::exec_policy_nosync(stream),
                                                 invalid_index_map->begin(),
                                                 invalid_index_map->end(),
                                                 cuda::std::identity());
  }
  return join_size + left_join_complement_size;
}
}  // namespace

template <typename Hasher>
hash_join<Hasher>::hash_join(cudf::table_view const& build,
                             bool has_nulls,
                             cudf::null_equality compare_nulls,
                             double load_factor,
                             rmm::cuda_stream_view stream)
  : _has_nulls(has_nulls),
    _is_empty{build.num_rows() == 0},
    _nulls_equal{compare_nulls},
    _hash_table{
      cuco::extent{static_cast<size_t>(build.num_rows())},
      load_factor,
      cuco::empty_key{
        cuco::pair{std::numeric_limits<hash_value_type>::max(), cudf::detail::JoinNoneValue}},
      {},
      {},
      {},
      {},
      cudf::detail::cuco_allocator<char>{rmm::mr::polymorphic_allocator<char>{}, stream.value()},
      stream.value()},
    _build{build},
    _preprocessed_build{
      cudf::experimental::row::equality::preprocessed_table::create(_build, stream)}
{
  CUDF_FUNC_RANGE();
  CUDF_EXPECTS(0 != build.num_columns(), "Hash join build table is empty", std::invalid_argument);
  CUDF_EXPECTS(load_factor > 0 && load_factor <= 1,
               "Invalid load factor: must be greater than 0 and less than or equal to 1.",
               std::invalid_argument);

  if (_is_empty) { return; }

  auto const row_bitmask =
    cudf::detail::bitmask_and(build, stream, cudf::get_current_device_resource_ref()).first;
  cudf::detail::build_hash_join(_build,
                                _preprocessed_build,
                                _hash_table,
                                _has_nulls,
                                _nulls_equal,
                                reinterpret_cast<bitmask_type const*>(row_bitmask.data()),
                                stream);
}

template <typename Hasher>
std::pair<std::unique_ptr<rmm::device_uvector<size_type>>,
          std::unique_ptr<rmm::device_uvector<size_type>>>
hash_join<Hasher>::inner_join(cudf::table_view const& probe,
                              std::optional<std::size_t> output_size,
                              rmm::cuda_stream_view stream,
                              rmm::device_async_resource_ref mr) const
{
  CUDF_FUNC_RANGE();
  return compute_hash_join(probe, cudf::detail::join_kind::INNER_JOIN, output_size, stream, mr);
}

template <typename Hasher>
std::pair<std::unique_ptr<rmm::device_uvector<size_type>>,
          std::unique_ptr<rmm::device_uvector<size_type>>>
hash_join<Hasher>::left_join(cudf::table_view const& probe,
                             std::optional<std::size_t> output_size,
                             rmm::cuda_stream_view stream,
                             rmm::device_async_resource_ref mr) const
{
  CUDF_FUNC_RANGE();
  return compute_hash_join(probe, cudf::detail::join_kind::LEFT_JOIN, output_size, stream, mr);
}

template <typename Hasher>
std::pair<std::unique_ptr<rmm::device_uvector<size_type>>,
          std::unique_ptr<rmm::device_uvector<size_type>>>
hash_join<Hasher>::full_join(cudf::table_view const& probe,
                             std::optional<std::size_t> output_size,
                             rmm::cuda_stream_view stream,
                             rmm::device_async_resource_ref mr) const
{
  CUDF_FUNC_RANGE();
  return compute_hash_join(probe, cudf::detail::join_kind::FULL_JOIN, output_size, stream, mr);
}

template <typename Hasher>
std::size_t hash_join<Hasher>::inner_join_size(cudf::table_view const& probe,
                                               rmm::cuda_stream_view stream) const
{
  CUDF_FUNC_RANGE();

  // Return directly if build table is empty
  if (_is_empty) { return 0; }

  CUDF_EXPECTS(_has_nulls || !cudf::has_nested_nulls(probe),
               "Probe table has nulls while build table was not hashed with null check.",
               std::invalid_argument);

  auto const preprocessed_probe =
    cudf::experimental::row::equality::preprocessed_table::create(probe, stream);

  return cudf::detail::compute_join_output_size(_build,
                                                probe,
                                                _preprocessed_build,
                                                preprocessed_probe,
                                                _hash_table,
                                                cudf::detail::join_kind::INNER_JOIN,
                                                _has_nulls,
                                                _nulls_equal,
                                                stream);
}

template <typename Hasher>
std::size_t hash_join<Hasher>::left_join_size(cudf::table_view const& probe,
                                              rmm::cuda_stream_view stream) const
{
  CUDF_FUNC_RANGE();

  // Trivial left join case - exit early
  if (_is_empty) { return probe.num_rows(); }

  CUDF_EXPECTS(_has_nulls || !cudf::has_nested_nulls(probe),
               "Probe table has nulls while build table was not hashed with null check.",
               std::invalid_argument);

  auto const preprocessed_probe =
    cudf::experimental::row::equality::preprocessed_table::create(probe, stream);

  return cudf::detail::compute_join_output_size(_build,
                                                probe,
                                                _preprocessed_build,
                                                preprocessed_probe,
                                                _hash_table,
                                                cudf::detail::join_kind::LEFT_JOIN,
                                                _has_nulls,
                                                _nulls_equal,
                                                stream);
}

template <typename Hasher>
std::size_t hash_join<Hasher>::full_join_size(cudf::table_view const& probe,
                                              rmm::cuda_stream_view stream,
                                              rmm::device_async_resource_ref mr) const
{
  CUDF_FUNC_RANGE();

  // Trivial left join case - exit early
  if (_is_empty) { return probe.num_rows(); }

  CUDF_EXPECTS(_has_nulls || !cudf::has_nested_nulls(probe),
               "Probe table has nulls while build table was not hashed with null check.",
               std::invalid_argument);

  auto const preprocessed_probe =
    cudf::experimental::row::equality::preprocessed_table::create(probe, stream);

  return cudf::detail::get_full_join_size(_build,
                                          probe,
                                          _preprocessed_build,
                                          preprocessed_probe,
                                          _hash_table,
                                          _has_nulls,
                                          _nulls_equal,
                                          stream,
                                          mr);
}

template <typename Hasher>
template <typename OutputIterator>
void hash_join<Hasher>::compute_match_counts(cudf::table_view const& probe,
                                             OutputIterator output_iter,
                                             rmm::cuda_stream_view stream) const
{
  CUDF_EXPECTS(_has_nulls || !cudf::has_nested_nulls(probe),
               "Probe table has nulls while build table was not hashed with null check.",
               std::invalid_argument);

  auto const preprocessed_probe =
    cudf::experimental::row::equality::preprocessed_table::create(probe, stream);
  auto const probe_nulls          = cudf::nullate::DYNAMIC{_has_nulls};
  auto const probe_table_num_rows = probe.num_rows();

  auto compute_counts = [&](auto equality, auto d_hasher) {
    auto const iter = cudf::detail::make_counting_transform_iterator(0, pair_fn{d_hasher});
    _hash_table.count_each(iter,
                           iter + probe_table_num_rows,
                           equality,
                           _hash_table.hash_function(),
                           output_iter,
                           stream.value());
  };

  if (cudf::is_primitive_row_op_compatible(_build)) {
    auto const d_hasher = cudf::row::primitive::row_hasher{probe_nulls, preprocessed_probe};
    auto const d_equal  = cudf::row::primitive::row_equality_comparator{
      probe_nulls, preprocessed_probe, _preprocessed_build, _nulls_equal};
    compute_counts(primitive_pair_equal{d_equal}, d_hasher);
  } else {
    auto const d_hasher =
      cudf::experimental::row::hash::row_hasher{preprocessed_probe}.device_hasher(probe_nulls);
    auto const row_comparator = cudf::experimental::row::equality::two_table_comparator{
      preprocessed_probe, _preprocessed_build};
    auto const d_equal = row_comparator.equal_to<false>(probe_nulls, _nulls_equal);
    compute_counts(pair_equal{d_equal}, d_hasher);
  }
}

template <typename Hasher>
cudf::join_match_context hash_join<Hasher>::inner_join_match_context(
  cudf::table_view const& probe,
  rmm::cuda_stream_view stream,
  rmm::device_async_resource_ref mr) const
{
  cudf::scoped_range range{"hash_join::inner_join_match_context"};

  auto match_counts =
    std::make_unique<rmm::device_uvector<size_type>>(probe.num_rows(), stream, mr);

  if (_is_empty) {
    thrust::fill(rmm::exec_policy_nosync(stream), match_counts->begin(), match_counts->end(), 0);
  } else {
    compute_match_counts(probe, match_counts->begin(), stream);
  }

  return cudf::join_match_context{probe, std::move(match_counts)};
}

template <typename Hasher>
cudf::join_match_context hash_join<Hasher>::left_join_match_context(
  cudf::table_view const& probe,
  rmm::cuda_stream_view stream,
  rmm::device_async_resource_ref mr) const
{
  cudf::scoped_range range{"hash_join::left_join_match_context"};

  auto match_counts =
    std::make_unique<rmm::device_uvector<size_type>>(probe.num_rows(), stream, mr);

  if (_is_empty) {
    thrust::fill(rmm::exec_policy_nosync(stream), match_counts->begin(), match_counts->end(), 1);
  } else {
    auto transform = [] __device__(size_type count) { return count == 0 ? 1 : count; };
    auto transformed_output =
      thrust::make_transform_output_iterator(match_counts->begin(), transform);
    compute_match_counts(probe, transformed_output, stream);
  }

  return cudf::join_match_context{probe, std::move(match_counts)};
}

template <typename Hasher>
cudf::join_match_context hash_join<Hasher>::full_join_match_context(
  cudf::table_view const& probe,
  rmm::cuda_stream_view stream,
  rmm::device_async_resource_ref mr) const
{
  cudf::scoped_range range{"hash_join::full_join_match_context"};

  auto match_counts =
    std::make_unique<rmm::device_uvector<size_type>>(probe.num_rows(), stream, mr);

  if (_is_empty) {
    thrust::fill(rmm::exec_policy_nosync(stream), match_counts->begin(), match_counts->end(), 1);
  } else {
    auto transform = [] __device__(size_type count) { return count == 0 ? 1 : count; };
    auto transformed_output =
      thrust::make_transform_output_iterator(match_counts->begin(), transform);
    compute_match_counts(probe, transformed_output, stream);
  }

  return cudf::join_match_context{probe, std::move(match_counts)};
}

template <typename Hasher>
std::pair<std::unique_ptr<rmm::device_uvector<size_type>>,
          std::unique_ptr<rmm::device_uvector<size_type>>>
hash_join<Hasher>::probe_join_indices(cudf::table_view const& probe_table,
                                      cudf::detail::join_kind join,
                                      std::optional<std::size_t> output_size,
                                      rmm::cuda_stream_view stream,
                                      rmm::device_async_resource_ref mr) const
{
  // Trivial left join case - exit early
  if (_is_empty and join != cudf::detail::join_kind::INNER_JOIN) {
    return get_trivial_left_join_indices(probe_table, stream, mr);
  }

  CUDF_EXPECTS(!_is_empty, "Hash table of hash join is null.");

  CUDF_EXPECTS(_has_nulls || !cudf::has_nested_nulls(probe_table),
               "Probe table has nulls while build table was not hashed with null check.",
               std::invalid_argument);

  auto const preprocessed_probe =
    cudf::experimental::row::equality::preprocessed_table::create(probe_table, stream);
  auto join_indices = cudf::detail::probe_join_hash_table(_build,
                                                          probe_table,
                                                          _preprocessed_build,
                                                          preprocessed_probe,
                                                          _hash_table,
                                                          join,
                                                          _has_nulls,
                                                          _nulls_equal,
                                                          output_size,
                                                          stream,
                                                          mr);

  if (join == cudf::detail::join_kind::FULL_JOIN) {
    auto complement_indices = detail::get_left_join_indices_complement(
      join_indices.second, probe_table.num_rows(), _build.num_rows(), stream, mr);
    join_indices = detail::concatenate_vector_pairs(join_indices, complement_indices, stream);
  }
  return join_indices;
}

template <typename Hasher>
std::pair<std::unique_ptr<rmm::device_uvector<size_type>>,
          std::unique_ptr<rmm::device_uvector<size_type>>>
hash_join<Hasher>::compute_hash_join(cudf::table_view const& probe,
                                     cudf::detail::join_kind join,
                                     std::optional<std::size_t> output_size,
                                     rmm::cuda_stream_view stream,
                                     rmm::device_async_resource_ref mr) const
{
  CUDF_EXPECTS(0 != probe.num_columns(), "Hash join probe table is empty", std::invalid_argument);

  CUDF_EXPECTS(_build.num_columns() == probe.num_columns(),
               "Mismatch in number of columns to be joined on",
               std::invalid_argument);

  CUDF_EXPECTS(_has_nulls || !cudf::has_nested_nulls(probe),
               "Probe table has nulls while build table was not hashed with null check.",
               std::invalid_argument);

  if (is_trivial_join(probe, _build, join)) {
    return std::pair(std::make_unique<rmm::device_uvector<size_type>>(0, stream, mr),
                     std::make_unique<rmm::device_uvector<size_type>>(0, stream, mr));
  }

  CUDF_EXPECTS(cudf::have_same_types(_build, probe),
               "Mismatch in joining column data types",
               cudf::data_type_error);

  return probe_join_indices(probe, join, output_size, stream, mr);
}
}  // namespace detail

hash_join::~hash_join() = default;

hash_join::hash_join(cudf::table_view const& build,
                     null_equality compare_nulls,
                     rmm::cuda_stream_view stream)
  : hash_join(
      build, nullable_join::YES, compare_nulls, cudf::detail::CUCO_DESIRED_LOAD_FACTOR, stream)
{
}

hash_join::hash_join(cudf::table_view const& build,
                     nullable_join has_nulls,
                     null_equality compare_nulls,
                     double load_factor,
                     rmm::cuda_stream_view stream)
  : _impl{std::make_unique<impl_type const>(
      build, has_nulls == nullable_join::YES, compare_nulls, load_factor, stream)}
{
}

std::pair<std::unique_ptr<rmm::device_uvector<size_type>>,
          std::unique_ptr<rmm::device_uvector<size_type>>>
hash_join::inner_join(cudf::table_view const& probe,
                      std::optional<std::size_t> output_size,
                      rmm::cuda_stream_view stream,
                      rmm::device_async_resource_ref mr) const
{
  return _impl->inner_join(probe, output_size, stream, mr);
}

std::pair<std::unique_ptr<rmm::device_uvector<size_type>>,
          std::unique_ptr<rmm::device_uvector<size_type>>>
hash_join::left_join(cudf::table_view const& probe,
                     std::optional<std::size_t> output_size,
                     rmm::cuda_stream_view stream,
                     rmm::device_async_resource_ref mr) const
{
  return _impl->left_join(probe, output_size, stream, mr);
}

std::pair<std::unique_ptr<rmm::device_uvector<size_type>>,
          std::unique_ptr<rmm::device_uvector<size_type>>>
hash_join::full_join(cudf::table_view const& probe,
                     std::optional<std::size_t> output_size,
                     rmm::cuda_stream_view stream,
                     rmm::device_async_resource_ref mr) const
{
  return _impl->full_join(probe, output_size, stream, mr);
}

std::size_t hash_join::inner_join_size(cudf::table_view const& probe,
                                       rmm::cuda_stream_view stream) const
{
  return _impl->inner_join_size(probe, stream);
}

std::size_t hash_join::left_join_size(cudf::table_view const& probe,
                                      rmm::cuda_stream_view stream) const
{
  return _impl->left_join_size(probe, stream);
}

std::size_t hash_join::full_join_size(cudf::table_view const& probe,
                                      rmm::cuda_stream_view stream,
                                      rmm::device_async_resource_ref mr) const
{
  return _impl->full_join_size(probe, stream, mr);
}

cudf::join_match_context hash_join::inner_join_match_context(
  cudf::table_view const& probe,
  rmm::cuda_stream_view stream,
  rmm::device_async_resource_ref mr) const
{
  return _impl->inner_join_match_context(probe, stream, mr);
}

cudf::join_match_context hash_join::left_join_match_context(cudf::table_view const& probe,
                                                            rmm::cuda_stream_view stream,
                                                            rmm::device_async_resource_ref mr) const
{
  return _impl->left_join_match_context(probe, stream, mr);
}

cudf::join_match_context hash_join::full_join_match_context(cudf::table_view const& probe,
                                                            rmm::cuda_stream_view stream,
                                                            rmm::device_async_resource_ref mr) const
{
  return _impl->full_join_match_context(probe, stream, mr);
}

}  // namespace cudf
