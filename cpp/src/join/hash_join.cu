#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2020-2024, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */
#include "join_common_utils.cuh"

#include <cudf/copying.hpp>
#include <cudf/detail/iterator.cuh>
#include <cudf/detail/join.hpp>
#include <cudf/detail/structs/utilities.hpp>
#include <cudf/hashing/detail/helper_functions.cuh>
#include <cudf/join.hpp>

#include <rmm/cuda_stream_view.hpp>
#include <rmm/device_buffer.hpp>
#include <rmm/device_uvector.hpp>
#include <rmm/exec_policy.hpp>
#include <rmm/resource_ref.hpp>

#include <thrust/count.h>
#include <thrust/functional.h>
#include <thrust/iterator/constant_iterator.h>
#include <thrust/iterator/discard_iterator.h>
#include <thrust/iterator/zip_iterator.h>
#include <thrust/scatter.h>
#include <thrust/tuple.h>
#include <thrust/uninitialized_fill.h>

#include <cstddef>
#include <iostream>
#include <numeric>

namespace cudf {
namespace detail {
namespace {
/**
 * @brief Calculates the exact size of the join output produced when
 * joining two tables together.
 *
 * @throw cudf::logic_error if join is not INNER_JOIN or LEFT_JOIN
 *
 * @param build_table The right hand table
 * @param probe_table The left hand table
 * @param preprocessed_build shared_ptr to cudf::experimental::row::equality::preprocessed_table for
 *                           build_table
 * @param preprocessed_probe shared_ptr to cudf::experimental::row::equality::preprocessed_table for
 *                           probe_table
 * @param hash_table A hash table built on the build table that maps the index
 *                   of every row to the hash value of that row
 * @param join The type of join to be performed
 * @param has_nulls Flag to denote if build or probe tables have nested nulls
 * @param nulls_equal Flag to denote nulls are equal or not
 * @param stream CUDA stream used for device memory operations and kernel launches
 *
 * @return The exact size of the output of the join operation
 */
std::size_t compute_join_output_size(
  table_view const& build_table,
  table_view const& probe_table,
  std::shared_ptr<cudf::experimental::row::equality::preprocessed_table> const& preprocessed_build,
  std::shared_ptr<cudf::experimental::row::equality::preprocessed_table> const& preprocessed_probe,
  cudf::detail::multimap_type const& hash_table,
  join_kind join,
  bool has_nulls,
  cudf::null_equality nulls_equal,
  rmm::cuda_stream_view stream)
{
  size_type const build_table_num_rows{build_table.num_rows()};
  size_type const probe_table_num_rows{probe_table.num_rows()};

  // If the build table is empty, we know exactly how large the output
  // will be for the different types of joins and can return immediately
  if (0 == build_table_num_rows) {
    switch (join) {
      // Inner join with an empty table will have no output
      case join_kind::INNER_JOIN: return 0;

      // Left join with an empty table will have an output of NULL rows
      // equal to the number of rows in the probe table
      case join_kind::LEFT_JOIN: return probe_table_num_rows;

      default: CUDF_FAIL("Unsupported join type");
    }
  }

  auto const probe_nulls = cudf::nullate::DYNAMIC{has_nulls};

  auto const row_hash           = cudf::experimental::row::hash::row_hasher{preprocessed_probe};
  auto const hash_probe         = row_hash.device_hasher(probe_nulls);
  auto const empty_key_sentinel = hash_table.get_empty_key_sentinel();
  auto const iter               = cudf::detail::make_counting_transform_iterator(
    0, make_pair_function{hash_probe, empty_key_sentinel});

  auto const row_comparator =
    cudf::experimental::row::equality::two_table_comparator{preprocessed_probe, preprocessed_build};
  auto const comparator_helper = [&](auto device_comparator) {
    pair_equality equality{device_comparator};

    if (join == join_kind::LEFT_JOIN) {
      return hash_table.pair_count_outer(
        iter, iter + probe_table_num_rows, equality, stream.value());
    } else {
      return hash_table.pair_count(iter, iter + probe_table_num_rows, equality, stream.value());
    }
  };

  if (cudf::detail::has_nested_columns(probe_table)) {
    auto const device_comparator = row_comparator.equal_to<true>(has_nulls, nulls_equal);
    return comparator_helper(device_comparator);
  } else {
    auto const device_comparator = row_comparator.equal_to<false>(has_nulls, nulls_equal);
    return comparator_helper(device_comparator);
  }
}

/**
 * @brief Probes the `hash_table` built from `build_table` for tuples in `probe_table`,
 * and returns the output indices of `build_table` and `probe_table` as a combined table.
 * Behavior is undefined if the provided `output_size` is smaller than the actual output size.
 *
 * @param build_table Table of build side columns to join
 * @param probe_table Table of probe side columns to join
 * @param preprocessed_build shared_ptr to cudf::experimental::row::equality::preprocessed_table for
 *                           build_table
 * @param preprocessed_probe shared_ptr to cudf::experimental::row::equality::preprocessed_table for
 *                           probe_table
 * @param hash_table Hash table built from `build_table`
 * @param join The type of join to be performed
 * @param has_nulls Flag to denote if build or probe tables have nested nulls
 * @param compare_nulls Controls whether null join-key values should match or not
 * @param output_size Optional value which allows users to specify the exact output size
 * @param stream CUDA stream used for device memory operations and kernel launches
 * @param mr Device memory resource used to allocate the returned vectors
 *
 * @return Join output indices vector pair.
 */
std::pair<std::unique_ptr<rmm::device_uvector<size_type>>,
          std::unique_ptr<rmm::device_uvector<size_type>>>
probe_join_hash_table(
  cudf::table_view const& build_table,
  cudf::table_view const& probe_table,
  std::shared_ptr<cudf::experimental::row::equality::preprocessed_table> const& preprocessed_build,
  std::shared_ptr<cudf::experimental::row::equality::preprocessed_table> const& preprocessed_probe,
  cudf::detail::multimap_type const& hash_table,
  join_kind join,
  bool has_nulls,
  null_equality compare_nulls,
  std::optional<std::size_t> output_size,
  rmm::cuda_stream_view stream,
  rmm::device_async_resource_ref mr)
{
  // Use the output size directly if provided. Otherwise, compute the exact output size
  auto const probe_join_type =
    (join == cudf::detail::join_kind::FULL_JOIN) ? cudf::detail::join_kind::LEFT_JOIN : join;

  std::size_t const join_size = output_size ? *output_size
                                            : compute_join_output_size(build_table,
                                                                       probe_table,
                                                                       preprocessed_build,
                                                                       preprocessed_probe,
                                                                       hash_table,
                                                                       probe_join_type,
                                                                       has_nulls,
                                                                       compare_nulls,
                                                                       stream);

  // If output size is zero, return immediately
  if (join_size == 0) {
    return std::pair(std::make_unique<rmm::device_uvector<size_type>>(0, stream, mr),
                     std::make_unique<rmm::device_uvector<size_type>>(0, stream, mr));
  }

  auto left_indices  = std::make_unique<rmm::device_uvector<size_type>>(join_size, stream, mr);
  auto right_indices = std::make_unique<rmm::device_uvector<size_type>>(join_size, stream, mr);

  auto const probe_nulls = cudf::nullate::DYNAMIC{has_nulls};

  auto const row_hash           = cudf::experimental::row::hash::row_hasher{preprocessed_probe};
  auto const hash_probe         = row_hash.device_hasher(probe_nulls);
  auto const empty_key_sentinel = hash_table.get_empty_key_sentinel();
  auto const iter               = cudf::detail::make_counting_transform_iterator(
    0, make_pair_function{hash_probe, empty_key_sentinel});

  cudf::size_type const probe_table_num_rows = probe_table.num_rows();

  auto const out1_zip_begin = thrust::make_zip_iterator(
    thrust::make_tuple(thrust::make_discard_iterator(), left_indices->begin()));
  auto const out2_zip_begin = thrust::make_zip_iterator(
    thrust::make_tuple(thrust::make_discard_iterator(), right_indices->begin()));

  auto const row_comparator =
    cudf::experimental::row::equality::two_table_comparator{preprocessed_probe, preprocessed_build};
  auto const comparator_helper = [&](auto device_comparator) {
    pair_equality equality{device_comparator};

    if (join == cudf::detail::join_kind::FULL_JOIN or join == cudf::detail::join_kind::LEFT_JOIN) {
      [[maybe_unused]] auto [out1_zip_end, out2_zip_end] =
        hash_table.pair_retrieve_outer(iter,
                                       iter + probe_table_num_rows,
                                       out1_zip_begin,
                                       out2_zip_begin,
                                       equality,
                                       stream.value());

      if (join == cudf::detail::join_kind::FULL_JOIN) {
        auto const actual_size = thrust::distance(out1_zip_begin, out1_zip_end);
        left_indices->resize(actual_size, stream);
        right_indices->resize(actual_size, stream);
      }
    } else {
      hash_table.pair_retrieve(iter,
                               iter + probe_table_num_rows,
                               out1_zip_begin,
                               out2_zip_begin,
                               equality,
                               stream.value());
    }
  };

  if (cudf::detail::has_nested_columns(probe_table)) {
    auto const device_comparator = row_comparator.equal_to<true>(probe_nulls, compare_nulls);
    comparator_helper(device_comparator);
  } else {
    auto const device_comparator = row_comparator.equal_to<false>(probe_nulls, compare_nulls);
    comparator_helper(device_comparator);
  }

  return std::pair(std::move(left_indices), std::move(right_indices));
}

/**
 * @brief Probes the `hash_table` built from `build_table` for tuples in `probe_table` twice,
 * and returns the output size of a full join operation between `build_table` and `probe_table`.
 * TODO: this is a temporary solution as part of `full_join_size`. To be refactored during
 * cuco integration.
 *
 * @param build_table Table of build side columns to join
 * @param probe_table Table of probe side columns to join
 * @param preprocessed_build shared_ptr to cudf::experimental::row::equality::preprocessed_table for
 *                           build_table
 * @param preprocessed_probe shared_ptr to cudf::experimental::row::equality::preprocessed_table for
 *                           probe_table
 * @param hash_table Hash table built from `build_table`
 * @param has_nulls Flag to denote if build or probe tables have nested nulls
 * @param compare_nulls Controls whether null join-key values should match or not
 * @param stream CUDA stream used for device memory operations and kernel launches
 * @param mr Device memory resource used to allocate the intermediate vectors
 *
 * @return Output size of full join.
 */
std::size_t get_full_join_size(
  cudf::table_view const& build_table,
  cudf::table_view const& probe_table,
  std::shared_ptr<cudf::experimental::row::equality::preprocessed_table> const& preprocessed_build,
  std::shared_ptr<cudf::experimental::row::equality::preprocessed_table> const& preprocessed_probe,
  cudf::detail::multimap_type const& hash_table,
  bool has_nulls,
  null_equality compare_nulls,
  rmm::cuda_stream_view stream,
  rmm::device_async_resource_ref mr)
{
  std::size_t join_size = compute_join_output_size(build_table,
                                                   probe_table,
                                                   preprocessed_build,
                                                   preprocessed_probe,
                                                   hash_table,
                                                   cudf::detail::join_kind::LEFT_JOIN,
                                                   has_nulls,
                                                   compare_nulls,
                                                   stream);

  // If output size is zero, return immediately
  if (join_size == 0) { return join_size; }

  auto left_indices  = std::make_unique<rmm::device_uvector<size_type>>(join_size, stream, mr);
  auto right_indices = std::make_unique<rmm::device_uvector<size_type>>(join_size, stream, mr);

  auto const probe_nulls = cudf::nullate::DYNAMIC{has_nulls};

  auto const row_hash           = cudf::experimental::row::hash::row_hasher{preprocessed_probe};
  auto const hash_probe         = row_hash.device_hasher(probe_nulls);
  auto const empty_key_sentinel = hash_table.get_empty_key_sentinel();
  auto const iter               = cudf::detail::make_counting_transform_iterator(
    0, make_pair_function{hash_probe, empty_key_sentinel});

  cudf::size_type const probe_table_num_rows = probe_table.num_rows();

  auto const out1_zip_begin = thrust::make_zip_iterator(
    thrust::make_tuple(thrust::make_discard_iterator(), left_indices->begin()));
  auto const out2_zip_begin = thrust::make_zip_iterator(
    thrust::make_tuple(thrust::make_discard_iterator(), right_indices->begin()));

  auto const row_comparator =
    cudf::experimental::row::equality::two_table_comparator{preprocessed_probe, preprocessed_build};
  auto const comparator_helper = [&](auto device_comparator) {
    pair_equality equality{device_comparator};
    hash_table.pair_retrieve_outer(
      iter, iter + probe_table_num_rows, out1_zip_begin, out2_zip_begin, equality, stream.value());
  };
  if (cudf::detail::has_nested_columns(probe_table)) {
    auto const device_comparator = row_comparator.equal_to<true>(probe_nulls, compare_nulls);
    comparator_helper(device_comparator);
  } else {
    auto const device_comparator = row_comparator.equal_to<false>(probe_nulls, compare_nulls);
    comparator_helper(device_comparator);
  }

  // Release intermediate memory allocation
  left_indices->resize(0, stream);

  auto const left_table_row_count  = probe_table.num_rows();
  auto const right_table_row_count = build_table.num_rows();

  std::size_t left_join_complement_size;

  // If left table is empty then all rows of the right table should be represented in the joined
  // indices.
  if (left_table_row_count == 0) {
    left_join_complement_size = right_table_row_count;
  } else {
    // Assume all the indices in invalid_index_map are invalid
    auto invalid_index_map =
      std::make_unique<rmm::device_uvector<size_type>>(right_table_row_count, stream);
    thrust::uninitialized_fill(
      rmm::exec_policy(stream), invalid_index_map->begin(), invalid_index_map->end(), int32_t{1});

    // Functor to check for index validity since left joins can create invalid indices
    valid_range<size_type> valid(0, right_table_row_count);

    // invalid_index_map[index_ptr[i]] = 0 for i = 0 to right_table_row_count
    // Thus specifying that those locations are valid
    thrust::scatter_if(rmm::exec_policy(stream),
                       thrust::make_constant_iterator(0),
                       thrust::make_constant_iterator(0) + right_indices->size(),
                       right_indices->begin(),      // Index locations
                       right_indices->begin(),      // Stencil - Check if index location is valid
                       invalid_index_map->begin(),  // Output indices
                       valid);                      // Stencil Predicate

    // Create list of indices that have been marked as invalid
    left_join_complement_size = thrust::count_if(rmm::exec_policy(stream),
                                                 invalid_index_map->begin(),
                                                 invalid_index_map->end(),
                                                 thrust::identity());
  }
  return join_size + left_join_complement_size;
}
}  // namespace

template <typename Hasher>
hash_join<Hasher>::hash_join(cudf::table_view const& build,
                             bool has_nulls,
                             cudf::null_equality compare_nulls,
                             rmm::cuda_stream_view stream)
  : _has_nulls(has_nulls),
    _is_empty{build.num_rows() == 0},
    _nulls_equal{compare_nulls},
    _hash_table{compute_hash_table_size(build.num_rows()),
                cuco::empty_key{std::numeric_limits<hash_value_type>::max()},
                cuco::empty_value{cudf::detail::JoinNoneValue},
                stream.value(),
                cudf::detail::cuco_allocator{stream}},
    _build{build},
    _preprocessed_build{
      cudf::experimental::row::equality::preprocessed_table::create(_build, stream)}
{
  CUDF_FUNC_RANGE();
  CUDF_EXPECTS(0 != build.num_columns(), "Hash join build table is empty");

  if (_is_empty) { return; }

  auto const row_bitmask =
    cudf::detail::bitmask_and(build, stream, rmm::mr::get_current_device_resource()).first;
  cudf::detail::build_join_hash_table(_build,
                                      _preprocessed_build,
                                      _hash_table,
                                      _has_nulls,
                                      _nulls_equal,
                                      reinterpret_cast<bitmask_type const*>(row_bitmask.data()),
                                      stream);
}

template <typename Hasher>
std::pair<std::unique_ptr<rmm::device_uvector<size_type>>,
          std::unique_ptr<rmm::device_uvector<size_type>>>
hash_join<Hasher>::inner_join(cudf::table_view const& probe,
                              std::optional<std::size_t> output_size,
                              rmm::cuda_stream_view stream,
                              rmm::device_async_resource_ref mr) const
{
  CUDF_FUNC_RANGE();
  return compute_hash_join(probe, cudf::detail::join_kind::INNER_JOIN, output_size, stream, mr);
}

template <typename Hasher>
std::pair<std::unique_ptr<rmm::device_uvector<size_type>>,
          std::unique_ptr<rmm::device_uvector<size_type>>>
hash_join<Hasher>::left_join(cudf::table_view const& probe,
                             std::optional<std::size_t> output_size,
                             rmm::cuda_stream_view stream,
                             rmm::device_async_resource_ref mr) const
{
  CUDF_FUNC_RANGE();
  return compute_hash_join(probe, cudf::detail::join_kind::LEFT_JOIN, output_size, stream, mr);
}

template <typename Hasher>
std::pair<std::unique_ptr<rmm::device_uvector<size_type>>,
          std::unique_ptr<rmm::device_uvector<size_type>>>
hash_join<Hasher>::full_join(cudf::table_view const& probe,
                             std::optional<std::size_t> output_size,
                             rmm::cuda_stream_view stream,
                             rmm::device_async_resource_ref mr) const
{
  CUDF_FUNC_RANGE();
  return compute_hash_join(probe, cudf::detail::join_kind::FULL_JOIN, output_size, stream, mr);
}

template <typename Hasher>
std::size_t hash_join<Hasher>::inner_join_size(cudf::table_view const& probe,
                                               rmm::cuda_stream_view stream) const
{
  CUDF_FUNC_RANGE();

  // Return directly if build table is empty
  if (_is_empty) { return 0; }

  CUDF_EXPECTS(_has_nulls || !cudf::has_nested_nulls(probe),
               "Probe table has nulls while build table was not hashed with null check.");

  auto const preprocessed_probe =
    cudf::experimental::row::equality::preprocessed_table::create(probe, stream);

  return cudf::detail::compute_join_output_size(_build,
                                                probe,
                                                _preprocessed_build,
                                                preprocessed_probe,
                                                _hash_table,
                                                cudf::detail::join_kind::INNER_JOIN,
                                                _has_nulls,
                                                _nulls_equal,
                                                stream);
}

template <typename Hasher>
std::size_t hash_join<Hasher>::left_join_size(cudf::table_view const& probe,
                                              rmm::cuda_stream_view stream) const
{
  CUDF_FUNC_RANGE();

  // Trivial left join case - exit early
  if (_is_empty) { return probe.num_rows(); }

  CUDF_EXPECTS(_has_nulls || !cudf::has_nested_nulls(probe),
               "Probe table has nulls while build table was not hashed with null check.");

  auto const preprocessed_probe =
    cudf::experimental::row::equality::preprocessed_table::create(probe, stream);

  return cudf::detail::compute_join_output_size(_build,
                                                probe,
                                                _preprocessed_build,
                                                preprocessed_probe,
                                                _hash_table,
                                                cudf::detail::join_kind::LEFT_JOIN,
                                                _has_nulls,
                                                _nulls_equal,
                                                stream);
}

template <typename Hasher>
std::size_t hash_join<Hasher>::full_join_size(cudf::table_view const& probe,
                                              rmm::cuda_stream_view stream,
                                              rmm::device_async_resource_ref mr) const
{
  CUDF_FUNC_RANGE();

  // Trivial left join case - exit early
  if (_is_empty) { return probe.num_rows(); }

  CUDF_EXPECTS(_has_nulls || !cudf::has_nested_nulls(probe),
               "Probe table has nulls while build table was not hashed with null check.");

  auto const preprocessed_probe =
    cudf::experimental::row::equality::preprocessed_table::create(probe, stream);

  return cudf::detail::get_full_join_size(_build,
                                          probe,
                                          _preprocessed_build,
                                          preprocessed_probe,
                                          _hash_table,
                                          _has_nulls,
                                          _nulls_equal,
                                          stream,
                                          mr);
}

template <typename Hasher>
std::pair<std::unique_ptr<rmm::device_uvector<size_type>>,
          std::unique_ptr<rmm::device_uvector<size_type>>>
hash_join<Hasher>::probe_join_indices(cudf::table_view const& probe_table,
                                      cudf::detail::join_kind join,
                                      std::optional<std::size_t> output_size,
                                      rmm::cuda_stream_view stream,
                                      rmm::device_async_resource_ref mr) const
{
  // Trivial left join case - exit early
  if (_is_empty and join != cudf::detail::join_kind::INNER_JOIN) {
    return get_trivial_left_join_indices(probe_table, stream, mr);
  }

  CUDF_EXPECTS(!_is_empty, "Hash table of hash join is null.");

  CUDF_EXPECTS(_has_nulls || !cudf::has_nested_nulls(probe_table),
               "Probe table has nulls while build table was not hashed with null check.");

  auto const preprocessed_probe =
    cudf::experimental::row::equality::preprocessed_table::create(probe_table, stream);
  auto join_indices = cudf::detail::probe_join_hash_table(_build,
                                                          probe_table,
                                                          _preprocessed_build,
                                                          preprocessed_probe,
                                                          _hash_table,
                                                          join,
                                                          _has_nulls,
                                                          _nulls_equal,
                                                          output_size,
                                                          stream,
                                                          mr);

  if (join == cudf::detail::join_kind::FULL_JOIN) {
    auto complement_indices = detail::get_left_join_indices_complement(
      join_indices.second, probe_table.num_rows(), _build.num_rows(), stream, mr);
    join_indices = detail::concatenate_vector_pairs(join_indices, complement_indices, stream);
  }
  return join_indices;
}

template <typename Hasher>
std::pair<std::unique_ptr<rmm::device_uvector<size_type>>,
          std::unique_ptr<rmm::device_uvector<size_type>>>
hash_join<Hasher>::compute_hash_join(cudf::table_view const& probe,
                                     cudf::detail::join_kind join,
                                     std::optional<std::size_t> output_size,
                                     rmm::cuda_stream_view stream,
                                     rmm::device_async_resource_ref mr) const
{
  CUDF_EXPECTS(0 != probe.num_columns(), "Hash join probe table is empty");

  CUDF_EXPECTS(_build.num_columns() == probe.num_columns(),
               "Mismatch in number of columns to be joined on");

  CUDF_EXPECTS(_has_nulls || !cudf::has_nested_nulls(probe),
               "Probe table has nulls while build table was not hashed with null check.");

  if (is_trivial_join(probe, _build, join)) {
    return std::pair(std::make_unique<rmm::device_uvector<size_type>>(0, stream, mr),
                     std::make_unique<rmm::device_uvector<size_type>>(0, stream, mr));
  }

  CUDF_EXPECTS(std::equal(std::cbegin(_build),
                          std::cend(_build),
                          std::cbegin(probe),
                          std::cend(probe),
                          [](auto const& b, auto const& p) { return b.type() == p.type(); }),
               "Mismatch in joining column data types");

  return probe_join_indices(probe, join, output_size, stream, mr);
}
}  // namespace detail

hash_join::~hash_join() = default;

hash_join::hash_join(cudf::table_view const& build,
                     null_equality compare_nulls,
                     rmm::cuda_stream_view stream)
  // If we cannot know beforehand about null existence then let's assume that there are nulls.
  : hash_join(build, nullable_join::YES, compare_nulls, stream)
{
}

hash_join::hash_join(cudf::table_view const& build,
                     nullable_join has_nulls,
                     null_equality compare_nulls,
                     rmm::cuda_stream_view stream)
  : _impl{std::make_unique<impl_type const>(
      build, has_nulls == nullable_join::YES, compare_nulls, stream)}
{
}

std::pair<std::unique_ptr<rmm::device_uvector<size_type>>,
          std::unique_ptr<rmm::device_uvector<size_type>>>
hash_join::inner_join(cudf::table_view const& probe,
                      std::optional<std::size_t> output_size,
                      rmm::cuda_stream_view stream,
                      rmm::device_async_resource_ref mr) const
{
  return _impl->inner_join(probe, output_size, stream, mr);
}

std::pair<std::unique_ptr<rmm::device_uvector<size_type>>,
          std::unique_ptr<rmm::device_uvector<size_type>>>
hash_join::left_join(cudf::table_view const& probe,
                     std::optional<std::size_t> output_size,
                     rmm::cuda_stream_view stream,
                     rmm::device_async_resource_ref mr) const
{
  return _impl->left_join(probe, output_size, stream, mr);
}

std::pair<std::unique_ptr<rmm::device_uvector<size_type>>,
          std::unique_ptr<rmm::device_uvector<size_type>>>
hash_join::full_join(cudf::table_view const& probe,
                     std::optional<std::size_t> output_size,
                     rmm::cuda_stream_view stream,
                     rmm::device_async_resource_ref mr) const
{
  return _impl->full_join(probe, output_size, stream, mr);
}

std::size_t hash_join::inner_join_size(cudf::table_view const& probe,
                                       rmm::cuda_stream_view stream) const
{
  return _impl->inner_join_size(probe, stream);
}

std::size_t hash_join::left_join_size(cudf::table_view const& probe,
                                      rmm::cuda_stream_view stream) const
{
  return _impl->left_join_size(probe, stream);
}

std::size_t hash_join::full_join_size(cudf::table_view const& probe,
                                      rmm::cuda_stream_view stream,
                                      rmm::device_async_resource_ref mr) const
{
  return _impl->full_join_size(probe, stream, mr);
}

}  // namespace cudf
