#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2020-2022, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */
#include "join_common_utils.cuh"

#include <cudf/copying.hpp>
#include <cudf/detail/concatenate.cuh>
#include <cudf/detail/iterator.cuh>
#include <cudf/detail/join.hpp>
#include <cudf/detail/structs/utilities.hpp>
#include <cudf/join.hpp>

#include <rmm/cuda_stream_view.hpp>
#include <rmm/device_buffer.hpp>
#include <rmm/device_uvector.hpp>
#include <rmm/exec_policy.hpp>

#include <thrust/count.h>
#include <thrust/functional.h>
#include <thrust/iterator/constant_iterator.h>
#include <thrust/iterator/discard_iterator.h>
#include <thrust/iterator/zip_iterator.h>
#include <thrust/scatter.h>
#include <thrust/tuple.h>
#include <thrust/uninitialized_fill.h>

#include <cstddef>
#include <iostream>
#include <numeric>

namespace cudf {
namespace detail {
namespace {
/**
 * @brief Calculates the exact size of the join output produced when
 * joining two tables together.
 *
 * @throw cudf::logic_error if JoinKind is not INNER_JOIN or LEFT_JOIN
 *
 * @tparam JoinKind The type of join to be performed
 *
 * @param build_table The right hand table
 * @param probe_table The left hand table
 * @param hash_table A hash table built on the build table that maps the index
 * of every row to the hash value of that row.
 * @param nulls_equal Flag to denote nulls are equal or not.
 * @param stream CUDA stream used for device memory operations and kernel launches
 *
 * @return The exact size of the output of the join operation
 */
template <join_kind JoinKind>
std::size_t compute_join_output_size(table_device_view build_table,
                                     table_device_view probe_table,
                                     cudf::detail::multimap_type const& hash_table,
                                     bool const has_nulls,
                                     cudf::null_equality const nulls_equal,
                                     rmm::cuda_stream_view stream)
{
  const size_type build_table_num_rows{build_table.num_rows()};
  const size_type probe_table_num_rows{probe_table.num_rows()};

  // If the build table is empty, we know exactly how large the output
  // will be for the different types of joins and can return immediately
  if (0 == build_table_num_rows) {
    switch (JoinKind) {
      // Inner join with an empty table will have no output
      case join_kind::INNER_JOIN: return 0;

      // Left join with an empty table will have an output of NULL rows
      // equal to the number of rows in the probe table
      case join_kind::LEFT_JOIN: return probe_table_num_rows;

      default: CUDF_FAIL("Unsupported join type");
    }
  }

  auto const probe_nulls = cudf::nullate::DYNAMIC{has_nulls};
  pair_equality equality{probe_table, build_table, probe_nulls, nulls_equal};

  row_hash hash_probe{probe_nulls, probe_table};
  auto const empty_key_sentinel = hash_table.get_empty_key_sentinel();
  make_pair_function pair_func{hash_probe, empty_key_sentinel};

  auto iter = cudf::detail::make_counting_transform_iterator(0, pair_func);

  std::size_t size;
  if constexpr (JoinKind == join_kind::LEFT_JOIN) {
    size = hash_table.pair_count_outer(iter, iter + probe_table_num_rows, equality, stream.value());
  } else {
    size = hash_table.pair_count(iter, iter + probe_table_num_rows, equality, stream.value());
  }

  return size;
}

/**
 * @brief Probes the `hash_table` built from `build_table` for tuples in `probe_table`,
 * and returns the output indices of `build_table` and `probe_table` as a combined table.
 * Behavior is undefined if the provided `output_size` is smaller than the actual output size.
 *
 * @tparam JoinKind The type of join to be performed.
 *
 * @param build_table Table of build side columns to join.
 * @param probe_table Table of probe side columns to join.
 * @param hash_table Hash table built from `build_table`.
 * @param compare_nulls Controls whether null join-key values should match or not.
 * @param output_size Optional value which allows users to specify the exact output size.
 * @param stream CUDA stream used for device memory operations and kernel launches.
 * @param mr Device memory resource used to allocate the returned vectors.
 *
 * @return Join output indices vector pair.
 */
template <join_kind JoinKind>
std::pair<std::unique_ptr<rmm::device_uvector<size_type>>,
          std::unique_ptr<rmm::device_uvector<size_type>>>
probe_join_hash_table(cudf::table_device_view build_table,
                      cudf::table_device_view probe_table,
                      cudf::detail::multimap_type const& hash_table,
                      bool has_nulls,
                      null_equality compare_nulls,
                      std::optional<std::size_t> output_size,
                      rmm::cuda_stream_view stream,
                      rmm::mr::device_memory_resource* mr)
{
  // Use the output size directly if provided. Otherwise, compute the exact output size
  constexpr cudf::detail::join_kind ProbeJoinKind = (JoinKind == cudf::detail::join_kind::FULL_JOIN)
                                                      ? cudf::detail::join_kind::LEFT_JOIN
                                                      : JoinKind;

  std::size_t const join_size =
    output_size ? *output_size
                : compute_join_output_size<ProbeJoinKind>(
                    build_table, probe_table, hash_table, has_nulls, compare_nulls, stream);

  // If output size is zero, return immediately
  if (join_size == 0) {
    return std::pair(std::make_unique<rmm::device_uvector<size_type>>(0, stream, mr),
                     std::make_unique<rmm::device_uvector<size_type>>(0, stream, mr));
  }

  auto left_indices  = std::make_unique<rmm::device_uvector<size_type>>(join_size, stream, mr);
  auto right_indices = std::make_unique<rmm::device_uvector<size_type>>(join_size, stream, mr);

  auto const probe_nulls = cudf::nullate::DYNAMIC{has_nulls};
  pair_equality equality{probe_table, build_table, probe_nulls, compare_nulls};

  row_hash hash_probe{probe_nulls, probe_table};
  auto const empty_key_sentinel = hash_table.get_empty_key_sentinel();
  make_pair_function pair_func{hash_probe, empty_key_sentinel};

  auto iter = cudf::detail::make_counting_transform_iterator(0, pair_func);

  const cudf::size_type probe_table_num_rows = probe_table.num_rows();

  auto out1_zip_begin = thrust::make_zip_iterator(
    thrust::make_tuple(thrust::make_discard_iterator(), left_indices->begin()));
  auto out2_zip_begin = thrust::make_zip_iterator(
    thrust::make_tuple(thrust::make_discard_iterator(), right_indices->begin()));

  if constexpr (JoinKind == cudf::detail::join_kind::FULL_JOIN or
                JoinKind == cudf::detail::join_kind::LEFT_JOIN) {
    [[maybe_unused]] auto [out1_zip_end, out2_zip_end] = hash_table.pair_retrieve_outer(
      iter, iter + probe_table_num_rows, out1_zip_begin, out2_zip_begin, equality, stream.value());

    if constexpr (JoinKind == cudf::detail::join_kind::FULL_JOIN) {
      auto const actual_size = out1_zip_end - out1_zip_begin;
      left_indices->resize(actual_size, stream);
      right_indices->resize(actual_size, stream);
    }
  } else {
    hash_table.pair_retrieve(
      iter, iter + probe_table_num_rows, out1_zip_begin, out2_zip_begin, equality, stream.value());
  }
  return std::pair(std::move(left_indices), std::move(right_indices));
}

/**
 * @brief Probes the `hash_table` built from `build_table` for tuples in `probe_table` twice,
 * and returns the output size of a full join operation between `build_table` and `probe_table`.
 * TODO: this is a temporary solution as part of `full_join_size`. To be refactored during
 * cuco integration.
 *
 * @param build_table Table of build side columns to join.
 * @param probe_table Table of probe side columns to join.
 * @param hash_table Hash table built from `build_table`.
 * @param compare_nulls Controls whether null join-key values should match or not.
 * @param stream CUDA stream used for device memory operations and kernel launches.
 * @param mr Device memory resource used to allocate the intermediate vectors.
 *
 * @return Output size of full join.
 */
std::size_t get_full_join_size(cudf::table_device_view build_table,
                               cudf::table_device_view probe_table,
                               cudf::detail::multimap_type const& hash_table,
                               bool const has_nulls,
                               null_equality const compare_nulls,
                               rmm::cuda_stream_view stream,
                               rmm::mr::device_memory_resource* mr)
{
  std::size_t join_size = compute_join_output_size<cudf::detail::join_kind::LEFT_JOIN>(
    build_table, probe_table, hash_table, has_nulls, compare_nulls, stream);

  // If output size is zero, return immediately
  if (join_size == 0) { return join_size; }

  auto left_indices  = std::make_unique<rmm::device_uvector<size_type>>(join_size, stream, mr);
  auto right_indices = std::make_unique<rmm::device_uvector<size_type>>(join_size, stream, mr);

  auto const probe_nulls = cudf::nullate::DYNAMIC{has_nulls};
  pair_equality equality{probe_table, build_table, probe_nulls, compare_nulls};

  row_hash hash_probe{probe_nulls, probe_table};
  auto const empty_key_sentinel = hash_table.get_empty_key_sentinel();
  make_pair_function pair_func{hash_probe, empty_key_sentinel};

  auto iter = cudf::detail::make_counting_transform_iterator(0, pair_func);

  const cudf::size_type probe_table_num_rows = probe_table.num_rows();

  auto out1_zip_begin = thrust::make_zip_iterator(
    thrust::make_tuple(thrust::make_discard_iterator(), left_indices->begin()));
  auto out2_zip_begin = thrust::make_zip_iterator(
    thrust::make_tuple(thrust::make_discard_iterator(), right_indices->begin()));

  hash_table.pair_retrieve_outer(
    iter, iter + probe_table_num_rows, out1_zip_begin, out2_zip_begin, equality, stream.value());

  // Release intermediate memory allocation
  left_indices->resize(0, stream);

  auto const left_table_row_count  = probe_table.num_rows();
  auto const right_table_row_count = build_table.num_rows();

  std::size_t left_join_complement_size;

  // If left table is empty then all rows of the right table should be represented in the joined
  // indices.
  if (left_table_row_count == 0) {
    left_join_complement_size = right_table_row_count;
  } else {
    // Assume all the indices in invalid_index_map are invalid
    auto invalid_index_map =
      std::make_unique<rmm::device_uvector<size_type>>(right_table_row_count, stream);
    thrust::uninitialized_fill(
      rmm::exec_policy(stream), invalid_index_map->begin(), invalid_index_map->end(), int32_t{1});

    // Functor to check for index validity since left joins can create invalid indices
    valid_range<size_type> valid(0, right_table_row_count);

    // invalid_index_map[index_ptr[i]] = 0 for i = 0 to right_table_row_count
    // Thus specifying that those locations are valid
    thrust::scatter_if(rmm::exec_policy(stream),
                       thrust::make_constant_iterator(0),
                       thrust::make_constant_iterator(0) + right_indices->size(),
                       right_indices->begin(),      // Index locations
                       right_indices->begin(),      // Stencil - Check if index location is valid
                       invalid_index_map->begin(),  // Output indices
                       valid);                      // Stencil Predicate

    // Create list of indices that have been marked as invalid
    left_join_complement_size = thrust::count_if(rmm::exec_policy(stream),
                                                 invalid_index_map->begin(),
                                                 invalid_index_map->end(),
                                                 thrust::identity());
  }
  return join_size + left_join_complement_size;
}
}  // namespace

template <typename Hasher>
hash_join<Hasher>::hash_join(cudf::table_view const& build,
                             cudf::null_equality compare_nulls,
                             rmm::cuda_stream_view stream)
  : _is_empty{build.num_rows() == 0},
    _composite_bitmask{cudf::detail::bitmask_and(build, stream).first},
    _nulls_equal{compare_nulls},
    _hash_table{compute_hash_table_size(build.num_rows()),
                cuco::sentinel::empty_key{std::numeric_limits<hash_value_type>::max()},
                cuco::sentinel::empty_value{cudf::detail::JoinNoneValue},
                stream.value(),
                detail::hash_table_allocator_type{default_allocator<char>{}, stream}}
{
  CUDF_FUNC_RANGE();
  CUDF_EXPECTS(0 != build.num_columns(), "Hash join build table is empty");
  CUDF_EXPECTS(build.num_rows() < cudf::detail::MAX_JOIN_SIZE,
               "Build column size is too big for hash join");

  // need to store off the owning structures for some of the views in _build
  _flattened_build_table = structs::detail::flatten_nested_columns(
    build, {}, {}, structs::detail::column_nullability::FORCE);
  _build = _flattened_build_table;

  if (_is_empty) { return; }

  cudf::detail::build_join_hash_table(_build,
                                      _hash_table,
                                      _nulls_equal,
                                      static_cast<bitmask_type const*>(_composite_bitmask.data()),
                                      stream);
}

template <typename Hasher>
std::pair<std::unique_ptr<rmm::device_uvector<size_type>>,
          std::unique_ptr<rmm::device_uvector<size_type>>>
hash_join<Hasher>::inner_join(cudf::table_view const& probe,
                              std::optional<std::size_t> output_size,
                              rmm::cuda_stream_view stream,
                              rmm::mr::device_memory_resource* mr) const
{
  CUDF_FUNC_RANGE();
  return compute_hash_join<cudf::detail::join_kind::INNER_JOIN>(probe, output_size, stream, mr);
}

template <typename Hasher>
std::pair<std::unique_ptr<rmm::device_uvector<size_type>>,
          std::unique_ptr<rmm::device_uvector<size_type>>>
hash_join<Hasher>::left_join(cudf::table_view const& probe,
                             std::optional<std::size_t> output_size,
                             rmm::cuda_stream_view stream,
                             rmm::mr::device_memory_resource* mr) const
{
  CUDF_FUNC_RANGE();
  return compute_hash_join<cudf::detail::join_kind::LEFT_JOIN>(probe, output_size, stream, mr);
}

template <typename Hasher>
std::pair<std::unique_ptr<rmm::device_uvector<size_type>>,
          std::unique_ptr<rmm::device_uvector<size_type>>>
hash_join<Hasher>::full_join(cudf::table_view const& probe,
                             std::optional<std::size_t> output_size,
                             rmm::cuda_stream_view stream,
                             rmm::mr::device_memory_resource* mr) const
{
  CUDF_FUNC_RANGE();
  return compute_hash_join<cudf::detail::join_kind::FULL_JOIN>(probe, output_size, stream, mr);
}

template <typename Hasher>
std::size_t hash_join<Hasher>::inner_join_size(cudf::table_view const& probe,
                                               rmm::cuda_stream_view stream) const
{
  CUDF_FUNC_RANGE();

  // Return directly if build table is empty
  if (_is_empty) { return 0; }

  auto flattened_probe = structs::detail::flatten_nested_columns(
    probe, {}, {}, structs::detail::column_nullability::FORCE);
  auto const flattened_probe_table = flattened_probe.flattened_columns();

  auto build_table_ptr           = cudf::table_device_view::create(_build, stream);
  auto flattened_probe_table_ptr = cudf::table_device_view::create(flattened_probe_table, stream);

  return cudf::detail::compute_join_output_size<cudf::detail::join_kind::INNER_JOIN>(
    *build_table_ptr,
    *flattened_probe_table_ptr,
    _hash_table,
    cudf::has_nulls(flattened_probe_table) | cudf::has_nulls(_build),
    _nulls_equal,
    stream);
}

template <typename Hasher>
std::size_t hash_join<Hasher>::left_join_size(cudf::table_view const& probe,
                                              rmm::cuda_stream_view stream) const
{
  CUDF_FUNC_RANGE();

  // Trivial left join case - exit early
  if (_is_empty) { return probe.num_rows(); }

  auto flattened_probe = structs::detail::flatten_nested_columns(
    probe, {}, {}, structs::detail::column_nullability::FORCE);
  auto const flattened_probe_table = flattened_probe.flattened_columns();

  auto build_table_ptr           = cudf::table_device_view::create(_build, stream);
  auto flattened_probe_table_ptr = cudf::table_device_view::create(flattened_probe_table, stream);

  return cudf::detail::compute_join_output_size<cudf::detail::join_kind::LEFT_JOIN>(
    *build_table_ptr,
    *flattened_probe_table_ptr,
    _hash_table,
    cudf::has_nulls(flattened_probe_table) | cudf::has_nulls(_build),
    _nulls_equal,
    stream);
}

template <typename Hasher>
std::size_t hash_join<Hasher>::full_join_size(cudf::table_view const& probe,
                                              rmm::cuda_stream_view stream,
                                              rmm::mr::device_memory_resource* mr) const
{
  CUDF_FUNC_RANGE();

  // Trivial left join case - exit early
  if (_is_empty) { return probe.num_rows(); }

  auto flattened_probe = structs::detail::flatten_nested_columns(
    probe, {}, {}, structs::detail::column_nullability::FORCE);
  auto const flattened_probe_table = flattened_probe.flattened_columns();

  auto build_table_ptr           = cudf::table_device_view::create(_build, stream);
  auto flattened_probe_table_ptr = cudf::table_device_view::create(flattened_probe_table, stream);

  return cudf::detail::get_full_join_size(
    *build_table_ptr,
    *flattened_probe_table_ptr,
    _hash_table,
    cudf::has_nulls(flattened_probe_table) | cudf::has_nulls(_build),
    _nulls_equal,
    stream,
    mr);
}

template <typename Hasher>
template <cudf::detail::join_kind JoinKind>
std::pair<std::unique_ptr<rmm::device_uvector<size_type>>,
          std::unique_ptr<rmm::device_uvector<size_type>>>
hash_join<Hasher>::probe_join_indices(cudf::table_view const& probe_table,
                                      std::optional<std::size_t> output_size,
                                      rmm::cuda_stream_view stream,
                                      rmm::mr::device_memory_resource* mr) const
{
  // Trivial left join case - exit early
  if (_is_empty and JoinKind != cudf::detail::join_kind::INNER_JOIN) {
    return get_trivial_left_join_indices(probe_table, stream, mr);
  }

  CUDF_EXPECTS(!_is_empty, "Hash table of hash join is null.");

  auto build_table_ptr = cudf::table_device_view::create(_build, stream);
  auto probe_table_ptr = cudf::table_device_view::create(probe_table, stream);

  auto join_indices = cudf::detail::probe_join_hash_table<JoinKind>(
    *build_table_ptr,
    *probe_table_ptr,
    _hash_table,
    cudf::has_nulls(probe_table) | cudf::has_nulls(_build),
    _nulls_equal,
    output_size,
    stream,
    mr);

  if constexpr (JoinKind == cudf::detail::join_kind::FULL_JOIN) {
    auto complement_indices = detail::get_left_join_indices_complement(
      join_indices.second, probe_table.num_rows(), _build.num_rows(), stream, mr);
    join_indices = detail::concatenate_vector_pairs(join_indices, complement_indices, stream);
  }
  return join_indices;
}

template <typename Hasher>
template <cudf::detail::join_kind JoinKind>
std::pair<std::unique_ptr<rmm::device_uvector<size_type>>,
          std::unique_ptr<rmm::device_uvector<size_type>>>
hash_join<Hasher>::compute_hash_join(cudf::table_view const& probe,
                                     std::optional<std::size_t> output_size,
                                     rmm::cuda_stream_view stream,
                                     rmm::mr::device_memory_resource* mr) const
{
  CUDF_EXPECTS(0 != probe.num_columns(), "Hash join probe table is empty");
  CUDF_EXPECTS(probe.num_rows() < cudf::detail::MAX_JOIN_SIZE,
               "Probe column size is too big for hash join");

  auto flattened_probe = structs::detail::flatten_nested_columns(
    probe, {}, {}, structs::detail::column_nullability::FORCE);
  auto const flattened_probe_table = flattened_probe.flattened_columns();

  CUDF_EXPECTS(_build.num_columns() == flattened_probe_table.num_columns(),
               "Mismatch in number of columns to be joined on");

  if (is_trivial_join(flattened_probe_table, _build, JoinKind)) {
    return std::pair(std::make_unique<rmm::device_uvector<size_type>>(0, stream, mr),
                     std::make_unique<rmm::device_uvector<size_type>>(0, stream, mr));
  }

  CUDF_EXPECTS(std::equal(std::cbegin(_build),
                          std::cend(_build),
                          std::cbegin(flattened_probe_table),
                          std::cend(flattened_probe_table),
                          [](const auto& b, const auto& p) { return b.type() == p.type(); }),
               "Mismatch in joining column data types");

  return probe_join_indices<JoinKind>(flattened_probe_table, output_size, stream, mr);
}
}  // namespace detail

hash_join::~hash_join() = default;

hash_join::hash_join(cudf::table_view const& build,
                     null_equality compare_nulls,
                     rmm::cuda_stream_view stream)
  : _impl{std::make_unique<const impl_type>(build, compare_nulls, stream)}
{
}

std::pair<std::unique_ptr<rmm::device_uvector<size_type>>,
          std::unique_ptr<rmm::device_uvector<size_type>>>
hash_join::inner_join(cudf::table_view const& probe,
                      std::optional<std::size_t> output_size,
                      rmm::cuda_stream_view stream,
                      rmm::mr::device_memory_resource* mr) const
{
  return _impl->inner_join(probe, output_size, stream, mr);
}

std::pair<std::unique_ptr<rmm::device_uvector<size_type>>,
          std::unique_ptr<rmm::device_uvector<size_type>>>
hash_join::left_join(cudf::table_view const& probe,
                     std::optional<std::size_t> output_size,
                     rmm::cuda_stream_view stream,
                     rmm::mr::device_memory_resource* mr) const
{
  return _impl->left_join(probe, output_size, stream, mr);
}

std::pair<std::unique_ptr<rmm::device_uvector<size_type>>,
          std::unique_ptr<rmm::device_uvector<size_type>>>
hash_join::full_join(cudf::table_view const& probe,
                     std::optional<std::size_t> output_size,
                     rmm::cuda_stream_view stream,
                     rmm::mr::device_memory_resource* mr) const
{
  return _impl->full_join(probe, output_size, stream, mr);
}

std::size_t hash_join::inner_join_size(cudf::table_view const& probe,
                                       rmm::cuda_stream_view stream) const
{
  return _impl->inner_join_size(probe, stream);
}

std::size_t hash_join::left_join_size(cudf::table_view const& probe,
                                      rmm::cuda_stream_view stream) const
{
  return _impl->left_join_size(probe, stream);
}

std::size_t hash_join::full_join_size(cudf::table_view const& probe,
                                      rmm::cuda_stream_view stream,
                                      rmm::mr::device_memory_resource* mr) const
{
  return _impl->full_join_size(probe, stream, mr);
}

}  // namespace cudf
