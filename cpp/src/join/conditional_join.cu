/*
 * Copyright (c) 2021-2024, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "join/conditional_join.hpp"
#include "join/conditional_join_kernels.cuh"
#include "join/join_common_utils.cuh"
#include "join/join_common_utils.hpp"

#include <cudf/ast/detail/expression_parser.hpp>
#include <cudf/ast/expressions.hpp>
#include <cudf/detail/utilities/cuda.cuh>
#include <cudf/join.hpp>
#include <cudf/table/table.hpp>
#include <cudf/table/table_device_view.cuh>
#include <cudf/table/table_view.hpp>
#include <cudf/types.hpp>
#include <cudf/utilities/memory_resource.hpp>

#include <rmm/cuda_stream_view.hpp>

#include <optional>

namespace cudf {
namespace detail {

std::unique_ptr<rmm::device_uvector<size_type>> conditional_join_anti_semi(
  table_view const& left,
  table_view const& right,
  ast::expression const& binary_predicate,
  join_kind join_type,
  std::optional<std::size_t> output_size,
  rmm::cuda_stream_view stream,
  rmm::device_async_resource_ref mr)
{
  if (right.num_rows() == 0) {
    switch (join_type) {
      case join_kind::LEFT_ANTI_JOIN: return get_trivial_left_join_indices(left, stream, mr).first;
      case join_kind::LEFT_SEMI_JOIN:
        return std::make_unique<rmm::device_uvector<size_type>>(0, stream, mr);
      default: CUDF_FAIL("Invalid join kind."); break;
    }
  } else if (left.num_rows() == 0) {
    switch (join_type) {
      case join_kind::LEFT_ANTI_JOIN: [[fallthrough]];
      case join_kind::LEFT_SEMI_JOIN:
        return std::make_unique<rmm::device_uvector<size_type>>(0, stream, mr);
      default: CUDF_FAIL("Invalid join kind."); break;
    }
  }

  auto const has_nulls = binary_predicate.may_evaluate_null(left, right, stream);

  auto const parser =
    ast::detail::expression_parser{binary_predicate, left, right, has_nulls, stream, mr};
  CUDF_EXPECTS(parser.output_type().id() == type_id::BOOL8,
               "The expression must produce a Boolean output.");

  auto left_table  = table_device_view::create(left, stream);
  auto right_table = table_device_view::create(right, stream);

  detail::grid_1d const config(left.num_rows(), DEFAULT_JOIN_BLOCK_SIZE);
  auto const shmem_size_per_block = parser.shmem_per_thread * config.num_threads_per_block;

  // TODO: Remove the output_size parameter. It is not needed because the
  // output size is bounded by the size of the left table.
  std::size_t join_size;
  if (output_size.has_value()) {
    join_size = *output_size;
  } else {
    // Allocate storage for the counter used to get the size of the join output
    rmm::device_scalar<std::size_t> size(0, stream, mr);
    if (has_nulls) {
      compute_conditional_join_output_size<DEFAULT_JOIN_BLOCK_SIZE, true>
        <<<config.num_blocks, config.num_threads_per_block, shmem_size_per_block, stream.value()>>>(
          *left_table, *right_table, join_type, parser.device_expression_data, false, size.data());
    } else {
      compute_conditional_join_output_size<DEFAULT_JOIN_BLOCK_SIZE, false>
        <<<config.num_blocks, config.num_threads_per_block, shmem_size_per_block, stream.value()>>>(
          *left_table, *right_table, join_type, parser.device_expression_data, false, size.data());
    }
    join_size = size.value(stream);
  }

  rmm::device_scalar<std::size_t> write_index(0, stream);

  auto left_indices = std::make_unique<rmm::device_uvector<size_type>>(join_size, stream, mr);

  auto const& join_output_l = left_indices->data();

  if (has_nulls) {
    conditional_join_anti_semi<DEFAULT_JOIN_BLOCK_SIZE, DEFAULT_JOIN_CACHE_SIZE, true>
      <<<config.num_blocks, config.num_threads_per_block, shmem_size_per_block, stream.value()>>>(
        *left_table,
        *right_table,
        join_type,
        join_output_l,
        write_index.data(),
        parser.device_expression_data,
        join_size);
  } else {
    conditional_join_anti_semi<DEFAULT_JOIN_BLOCK_SIZE, DEFAULT_JOIN_CACHE_SIZE, false>
      <<<config.num_blocks, config.num_threads_per_block, shmem_size_per_block, stream.value()>>>(
        *left_table,
        *right_table,
        join_type,
        join_output_l,
        write_index.data(),
        parser.device_expression_data,
        join_size);
  }
  return left_indices;
}

std::pair<std::unique_ptr<rmm::device_uvector<size_type>>,
          std::unique_ptr<rmm::device_uvector<size_type>>>
conditional_join(table_view const& left,
                 table_view const& right,
                 ast::expression const& binary_predicate,
                 join_kind join_type,
                 std::optional<std::size_t> output_size,
                 rmm::cuda_stream_view stream,
                 rmm::device_async_resource_ref mr)
{
  // We can immediately filter out cases where the right table is empty. In
  // some cases, we return all the rows of the left table with a corresponding
  // null index for the right table; in others, we return an empty output.
  if (right.num_rows() == 0) {
    switch (join_type) {
      // Left, left anti, and full all return all the row indices from left
      // with a corresponding NULL from the right.
      case join_kind::LEFT_JOIN:
      case join_kind::LEFT_ANTI_JOIN:
      case join_kind::FULL_JOIN: return get_trivial_left_join_indices(left, stream, mr);
      // Inner and left semi joins return empty output because no matches can exist.
      case join_kind::INNER_JOIN:
      case join_kind::LEFT_SEMI_JOIN:
        return std::pair(std::make_unique<rmm::device_uvector<size_type>>(0, stream, mr),
                         std::make_unique<rmm::device_uvector<size_type>>(0, stream, mr));
      default: CUDF_FAIL("Invalid join kind."); break;
    }
  } else if (left.num_rows() == 0) {
    switch (join_type) {
      // Left, left anti, left semi, and inner joins all return empty sets.
      case join_kind::LEFT_JOIN:
      case join_kind::LEFT_ANTI_JOIN:
      case join_kind::INNER_JOIN:
      case join_kind::LEFT_SEMI_JOIN:
        return std::pair(std::make_unique<rmm::device_uvector<size_type>>(0, stream, mr),
                         std::make_unique<rmm::device_uvector<size_type>>(0, stream, mr));
      // Full joins need to return the trivial complement.
      case join_kind::FULL_JOIN: {
        auto ret_flipped = get_trivial_left_join_indices(right, stream, mr);
        return std::pair(std::move(ret_flipped.second), std::move(ret_flipped.first));
      }
      default: CUDF_FAIL("Invalid join kind."); break;
    }
  }

  // If evaluating the expression may produce null outputs we create a nullable
  // output column and follow the null-supporting expression evaluation code
  // path.
  auto const has_nulls = binary_predicate.may_evaluate_null(left, right, stream);

  auto const parser =
    ast::detail::expression_parser{binary_predicate, left, right, has_nulls, stream, mr};
  CUDF_EXPECTS(parser.output_type().id() == type_id::BOOL8,
               "The expression must produce a boolean output.");

  auto left_table  = table_device_view::create(left, stream);
  auto right_table = table_device_view::create(right, stream);

  // For inner joins we support optimizing the join by launching one thread for
  // whichever table is larger rather than always using the left table.
  auto swap_tables = (join_type == join_kind::INNER_JOIN) && (right.num_rows() > left.num_rows());
  detail::grid_1d const config(swap_tables ? right.num_rows() : left.num_rows(),
                               DEFAULT_JOIN_BLOCK_SIZE);
  auto const shmem_size_per_block = parser.shmem_per_thread * config.num_threads_per_block;
  join_kind const kernel_join_type =
    join_type == join_kind::FULL_JOIN ? join_kind::LEFT_JOIN : join_type;

  // If the join size was not provided as an input, compute it here.
  std::size_t join_size;
  if (output_size.has_value()) {
    join_size = *output_size;
  } else {
    // Allocate storage for the counter used to get the size of the join output
    rmm::device_scalar<std::size_t> size(0, stream, mr);
    if (has_nulls) {
      compute_conditional_join_output_size<DEFAULT_JOIN_BLOCK_SIZE, true>
        <<<config.num_blocks, config.num_threads_per_block, shmem_size_per_block, stream.value()>>>(
          *left_table,
          *right_table,
          kernel_join_type,
          parser.device_expression_data,
          swap_tables,
          size.data());
    } else {
      compute_conditional_join_output_size<DEFAULT_JOIN_BLOCK_SIZE, false>
        <<<config.num_blocks, config.num_threads_per_block, shmem_size_per_block, stream.value()>>>(
          *left_table,
          *right_table,
          kernel_join_type,
          parser.device_expression_data,
          swap_tables,
          size.data());
    }
    join_size = size.value(stream);
  }

  // The initial early exit clauses guarantee that we will not reach this point
  // unless both the left and right tables are non-empty. Under that
  // constraint, neither left nor full joins can return an empty result since
  // at minimum we are guaranteed null matches for all non-matching rows. In
  // all other cases (inner, left semi, and left anti joins) if we reach this
  // point we can safely return an empty result.
  if (join_size == 0) {
    return std::pair(std::make_unique<rmm::device_uvector<size_type>>(0, stream, mr),
                     std::make_unique<rmm::device_uvector<size_type>>(0, stream, mr));
  }

  rmm::device_scalar<std::size_t> write_index(0, stream);

  auto left_indices  = std::make_unique<rmm::device_uvector<size_type>>(join_size, stream, mr);
  auto right_indices = std::make_unique<rmm::device_uvector<size_type>>(join_size, stream, mr);

  auto const& join_output_l = left_indices->data();
  auto const& join_output_r = right_indices->data();

  if (has_nulls) {
    conditional_join<DEFAULT_JOIN_BLOCK_SIZE, DEFAULT_JOIN_CACHE_SIZE, true>
      <<<config.num_blocks, config.num_threads_per_block, shmem_size_per_block, stream.value()>>>(
        *left_table,
        *right_table,
        kernel_join_type,
        join_output_l,
        join_output_r,
        write_index.data(),
        parser.device_expression_data,
        join_size,
        swap_tables);
  } else {
    conditional_join<DEFAULT_JOIN_BLOCK_SIZE, DEFAULT_JOIN_CACHE_SIZE, false>
      <<<config.num_blocks, config.num_threads_per_block, shmem_size_per_block, stream.value()>>>(
        *left_table,
        *right_table,
        kernel_join_type,
        join_output_l,
        join_output_r,
        write_index.data(),
        parser.device_expression_data,
        join_size,
        swap_tables);
  }

  auto join_indices = std::pair(std::move(left_indices), std::move(right_indices));

  // For full joins, get the indices in the right table that were not joined to
  // by any row in the left table.
  if (join_type == join_kind::FULL_JOIN) {
    auto complement_indices = detail::get_left_join_indices_complement(
      join_indices.second, left.num_rows(), right.num_rows(), stream, mr);
    join_indices = detail::concatenate_vector_pairs(join_indices, complement_indices, stream);
  }
  return join_indices;
}

std::size_t compute_conditional_join_output_size(table_view const& left,
                                                 table_view const& right,
                                                 ast::expression const& binary_predicate,
                                                 join_kind join_type,
                                                 rmm::cuda_stream_view stream,
                                                 rmm::device_async_resource_ref mr)
{
  // Until we add logic to handle the number of non-matches in the right table,
  // full joins are not supported in this function. Note that this does not
  // prevent actually performing full joins since we do that by calculating the
  // left join and then concatenating the complementary right indices.
  CUDF_EXPECTS(join_type != join_kind::FULL_JOIN,
               "Size estimation is not available for full joins.");

  // We can immediately filter out cases where one table is empty. In
  // some cases, we return all the rows of the other table with a corresponding
  // null index for the empty table; in others, we return an empty output.
  if (right.num_rows() == 0) {
    switch (join_type) {
      // Left, left anti, and full all return all the row indices from left
      // with a corresponding NULL from the right.
      case join_kind::LEFT_JOIN:
      case join_kind::LEFT_ANTI_JOIN:
      case join_kind::FULL_JOIN: return left.num_rows();
      // Inner and left semi joins return empty output because no matches can exist.
      case join_kind::INNER_JOIN:
      case join_kind::LEFT_SEMI_JOIN: return 0;
      default: CUDF_FAIL("Invalid join kind."); break;
    }
  } else if (left.num_rows() == 0) {
    switch (join_type) {
      // Left, left anti, left semi, and inner joins all return empty sets.
      case join_kind::LEFT_JOIN:
      case join_kind::LEFT_ANTI_JOIN:
      case join_kind::INNER_JOIN:
      case join_kind::LEFT_SEMI_JOIN: return 0;
      // Full joins need to return the trivial complement.
      case join_kind::FULL_JOIN: return right.num_rows();
      default: CUDF_FAIL("Invalid join kind."); break;
    }
  }

  // Prepare output column. Whether or not the output column is nullable is
  // determined by whether any of the columns in the input table are nullable.
  // If none of the input columns actually contain nulls, we can still use the
  // non-nullable version of the expression evaluation code path for
  // performance, so we capture that information as well.
  auto const has_nulls = binary_predicate.may_evaluate_null(left, right, stream);

  auto const parser =
    ast::detail::expression_parser{binary_predicate, left, right, has_nulls, stream, mr};
  CUDF_EXPECTS(parser.output_type().id() == type_id::BOOL8,
               "The expression must produce a boolean output.");

  auto left_table  = table_device_view::create(left, stream);
  auto right_table = table_device_view::create(right, stream);

  // For inner joins we support optimizing the join by launching one thread for
  // whichever table is larger rather than always using the left table.
  auto swap_tables = (join_type == join_kind::INNER_JOIN) && (right.num_rows() > left.num_rows());
  detail::grid_1d const config(swap_tables ? right.num_rows() : left.num_rows(),
                               DEFAULT_JOIN_BLOCK_SIZE);
  auto const shmem_size_per_block = parser.shmem_per_thread * config.num_threads_per_block;

  // Allocate storage for the counter used to get the size of the join output
  rmm::device_scalar<std::size_t> size(0, stream, mr);

  // Determine number of output rows without actually building the output to simply
  // find what the size of the output will be.
  if (has_nulls) {
    compute_conditional_join_output_size<DEFAULT_JOIN_BLOCK_SIZE, true>
      <<<config.num_blocks, config.num_threads_per_block, shmem_size_per_block, stream.value()>>>(
        *left_table,
        *right_table,
        join_type,
        parser.device_expression_data,
        swap_tables,
        size.data());
  } else {
    compute_conditional_join_output_size<DEFAULT_JOIN_BLOCK_SIZE, false>
      <<<config.num_blocks, config.num_threads_per_block, shmem_size_per_block, stream.value()>>>(
        *left_table,
        *right_table,
        join_type,
        parser.device_expression_data,
        swap_tables,
        size.data());
  }
  return size.value(stream);
}

}  // namespace detail

std::pair<std::unique_ptr<rmm::device_uvector<size_type>>,
          std::unique_ptr<rmm::device_uvector<size_type>>>
conditional_inner_join(table_view const& left,
                       table_view const& right,
                       ast::expression const& binary_predicate,
                       std::optional<std::size_t> output_size,
                       rmm::cuda_stream_view stream,
                       rmm::device_async_resource_ref mr)
{
  CUDF_FUNC_RANGE();
  return detail::conditional_join(
    left, right, binary_predicate, detail::join_kind::INNER_JOIN, output_size, stream, mr);
}

std::pair<std::unique_ptr<rmm::device_uvector<size_type>>,
          std::unique_ptr<rmm::device_uvector<size_type>>>
conditional_left_join(table_view const& left,
                      table_view const& right,
                      ast::expression const& binary_predicate,
                      std::optional<std::size_t> output_size,
                      rmm::cuda_stream_view stream,
                      rmm::device_async_resource_ref mr)
{
  CUDF_FUNC_RANGE();
  return detail::conditional_join(
    left, right, binary_predicate, detail::join_kind::LEFT_JOIN, output_size, stream, mr);
}

std::pair<std::unique_ptr<rmm::device_uvector<size_type>>,
          std::unique_ptr<rmm::device_uvector<size_type>>>
conditional_full_join(table_view const& left,
                      table_view const& right,
                      ast::expression const& binary_predicate,
                      rmm::cuda_stream_view stream,
                      rmm::device_async_resource_ref mr)
{
  CUDF_FUNC_RANGE();
  return detail::conditional_join(
    left, right, binary_predicate, detail::join_kind::FULL_JOIN, {}, stream, mr);
}

std::unique_ptr<rmm::device_uvector<size_type>> conditional_left_semi_join(
  table_view const& left,
  table_view const& right,
  ast::expression const& binary_predicate,
  std::optional<std::size_t> output_size,
  rmm::cuda_stream_view stream,
  rmm::device_async_resource_ref mr)
{
  CUDF_FUNC_RANGE();
  return detail::conditional_join_anti_semi(
    left, right, binary_predicate, detail::join_kind::LEFT_SEMI_JOIN, output_size, stream, mr);
}

std::unique_ptr<rmm::device_uvector<size_type>> conditional_left_anti_join(
  table_view const& left,
  table_view const& right,
  ast::expression const& binary_predicate,
  std::optional<std::size_t> output_size,
  rmm::cuda_stream_view stream,
  rmm::device_async_resource_ref mr)
{
  CUDF_FUNC_RANGE();
  return detail::conditional_join_anti_semi(
    left, right, binary_predicate, detail::join_kind::LEFT_ANTI_JOIN, output_size, stream, mr);
}

std::size_t conditional_inner_join_size(table_view const& left,
                                        table_view const& right,
                                        ast::expression const& binary_predicate,
                                        rmm::cuda_stream_view stream,
                                        rmm::device_async_resource_ref mr)
{
  CUDF_FUNC_RANGE();
  return detail::compute_conditional_join_output_size(
    left, right, binary_predicate, detail::join_kind::INNER_JOIN, stream, mr);
}

std::size_t conditional_left_join_size(table_view const& left,
                                       table_view const& right,
                                       ast::expression const& binary_predicate,
                                       rmm::cuda_stream_view stream,
                                       rmm::device_async_resource_ref mr)
{
  CUDF_FUNC_RANGE();
  return detail::compute_conditional_join_output_size(
    left, right, binary_predicate, detail::join_kind::LEFT_JOIN, stream, mr);
}

std::size_t conditional_left_semi_join_size(table_view const& left,
                                            table_view const& right,
                                            ast::expression const& binary_predicate,
                                            rmm::cuda_stream_view stream,
                                            rmm::device_async_resource_ref mr)
{
  CUDF_FUNC_RANGE();
  return detail::compute_conditional_join_output_size(
    left, right, binary_predicate, detail::join_kind::LEFT_SEMI_JOIN, stream, mr);
}

std::size_t conditional_left_anti_join_size(table_view const& left,
                                            table_view const& right,
                                            ast::expression const& binary_predicate,
                                            rmm::cuda_stream_view stream,
                                            rmm::device_async_resource_ref mr)
{
  CUDF_FUNC_RANGE();
  return detail::compute_conditional_join_output_size(
    left, right, binary_predicate, detail::join_kind::LEFT_ANTI_JOIN, stream, mr);
}

}  // namespace cudf
