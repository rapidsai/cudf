#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2017, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */


#include <limits>
#include <set>
#include <vector>

#include "cudf.h"
#include "rmm/rmm.h"
#include "utilities/error_utils.h"
#include "dataframe/cudf_table.cuh"
#include "utilities/nvtx/nvtx_utils.h"

#include "joining.h"

using namespace mgpu;

// Size limit due to use of int32 as join output.
// FIXME: upgrade to 64-bit
using output_index_type = int;
constexpr output_index_type MAX_JOIN_SIZE{std::numeric_limits<output_index_type>::max()};

/* --------------------------------------------------------------------------*/
/** 
 * @Synopsis Computes the Join result between two tables using the hash-based implementation. 
 * 
 * @Param num_cols The number of columns to join
 * @Param leftcol The left set of columns to join
 * @Param rightcol The right set of columns to join
 * @Param l_result The join computed indices of the left table
 * @Param r_result The join computed indices of the right table
 * @tparam join_type The type of join to be performed
 * @tparam size_type The data type used for size calculations
 * 
 * @Returns Upon successful computation, returns GDF_SUCCESS. Otherwise returns appropriate error code 
 */
/* ----------------------------------------------------------------------------*/
template <JoinType join_type, 
          typename size_type>
gdf_error hash_join(size_type num_cols, gdf_column **leftcol, gdf_column **rightcol,
                    gdf_column *l_result, gdf_column *r_result)
{
  // Wrap the set of gdf_columns in a gdf_table class
  std::unique_ptr< gdf_table<size_type> > left_table(new gdf_table<size_type>(num_cols, leftcol));
  std::unique_ptr< gdf_table<size_type> > right_table(new gdf_table<size_type>(num_cols, rightcol));

  return join_hash<join_type, output_index_type>(*left_table, 
                                                        *right_table, 
                                                        l_result, 
                                                        r_result);
}

template <JoinType join_type>
struct SortJoin {
template<typename launch_arg_t = mgpu::empty_t,
  typename a_it, typename b_it, typename comp_t>
    std::pair<gdf_column, gdf_column>
    operator()(a_it a, int a_count, b_it b, int b_count,
               comp_t comp, context_t& context) {
        return std::pair<gdf_column, gdf_column>();
    }
};

template <>
struct SortJoin<JoinType::INNER_JOIN> {
template<typename launch_arg_t = mgpu::empty_t,
  typename a_it, typename b_it, typename comp_t>
    std::pair<gdf_column, gdf_column>
    operator()(a_it a, int a_count, b_it b, int b_count,
               comp_t comp, context_t& context) {
        return inner_join(a, a_count, b, b_count, comp, context);
    }
};

template <>
struct SortJoin<JoinType::LEFT_JOIN> {
  template<typename launch_arg_t = mgpu::empty_t,
    typename a_it, typename b_it, typename comp_t>
    std::pair<gdf_column, gdf_column>
    operator()(a_it a, int a_count, b_it b, int b_count,
               comp_t comp, context_t& context) {
        return left_join(a, a_count, b, b_count, comp, context);
      }
};

template <JoinType join_type, typename T>
gdf_error sort_join_typed(gdf_column *leftcol, gdf_column *rightcol,
                          gdf_column *left_result, gdf_column *right_result,
                          gdf_context *ctxt) 
{
  using namespace mgpu;
  gdf_error err = GDF_SUCCESS;
  GDF_REQUIRE(!leftcol->valid  || !leftcol->null_count , GDF_VALIDITY_UNSUPPORTED);
  GDF_REQUIRE(!rightcol->valid || !rightcol->null_count, GDF_VALIDITY_UNSUPPORTED);

  rmm_mgpu_context_t context(false);
  SortJoin<join_type> sort_based_join;
  auto output = sort_based_join(static_cast<T*>(leftcol->data), leftcol->size,
                                       static_cast<T*>(rightcol->data), rightcol->size,
                                       less_t<T>(), context);
  *left_result = output.first;
  *right_result = output.second;
  CUDA_CHECK_LAST();

  return err;
}

/* --------------------------------------------------------------------------*/
/** 
 * @Synopsis  Computes the join operation between a single left and single right column
 using the sort based implementation.
 * 
 * @Param leftcol The left column to join
 * @Param rightcol The right column to join
 * @Param left_result The join computed indices of the left table
 * @Param right_result The join computed indices of the right table
 * @Param ctxt Structure that determines various run parameters, such as if the inputs
 are already sorted.
   @tparama join_type The type of join to perform
 * 
 * @Returns GDF_SUCCESS upon succesful completion of the join, otherwise returns 
 appropriate error code.
 */
/* ----------------------------------------------------------------------------*/
template <JoinType join_type>
gdf_error sort_join(gdf_column *leftcol, gdf_column *rightcol,
                    gdf_column *l_result, gdf_column *r_result,
                    gdf_context *ctxt)
{

  if(GDF_SORT != ctxt->flag_method) return GDF_INVALID_API_CALL;

  switch ( leftcol->dtype ){
    case GDF_INT8:      return sort_join_typed<join_type, int8_t>(leftcol, rightcol, l_result, r_result, ctxt);
    case GDF_INT16:     return sort_join_typed<join_type,int16_t>(leftcol, rightcol, l_result, r_result, ctxt);
    case GDF_INT32:     return sort_join_typed<join_type,int32_t>(leftcol, rightcol, l_result, r_result, ctxt);
    case GDF_INT64:     return sort_join_typed<join_type,int64_t>(leftcol, rightcol, l_result, r_result, ctxt);
    case GDF_FLOAT32:   return sort_join_typed<join_type,int32_t>(leftcol, rightcol, l_result, r_result, ctxt);
    case GDF_FLOAT64:   return sort_join_typed<join_type,int64_t>(leftcol, rightcol, l_result, r_result, ctxt);
    case GDF_DATE32:    return sort_join_typed<join_type,int32_t>(leftcol, rightcol, l_result, r_result, ctxt);
    case GDF_DATE64:    return sort_join_typed<join_type,int64_t>(leftcol, rightcol, l_result, r_result, ctxt);
    case GDF_TIMESTAMP: return sort_join_typed<join_type,int64_t>(leftcol, rightcol, l_result, r_result, ctxt);
    default: return GDF_UNSUPPORTED_DTYPE;
  }
}

template
gdf_error sort_join<JoinType::INNER_JOIN>(gdf_column *leftcol, gdf_column *rightcol,
                                          gdf_column *l_result, gdf_column *r_result,
                                          gdf_context *ctxt);
template
gdf_error sort_join<JoinType::LEFT_JOIN>(gdf_column *leftcol, gdf_column *rightcol,
                                         gdf_column *l_result, gdf_column *r_result,
                                         gdf_context *ctxt);

/* --------------------------------------------------------------------------*/
/**
* @Synopsis  Allocates a buffer and fills it with a repeated value
*
* @Param buffer Address of the buffer to be allocated
* @Param buffer_length Amount of memory to be allocated
* @Param value The value to be filled into the buffer
* @tparam data_type The data type to be used for the buffer
* @tparam size_type The data type used for size calculations
*/
/* ----------------------------------------------------------------------------*/
template <typename data_type,
          typename size_type>
gdf_error allocValueBuffer(data_type ** buffer,
                           const size_type buffer_length,
                           const data_type value) 
{
    RMM_TRY( RMM_ALLOC((void**)buffer, buffer_length*sizeof(data_type), 0) );
    thrust::fill(thrust::device, *buffer, *buffer + buffer_length, value);
    return GDF_SUCCESS;
}

/* --------------------------------------------------------------------------*/
/**
* @Synopsis  Allocates a buffer and fills it with a sequence
*
* @Param buffer Address of the buffer to be allocated
* @Param buffer_length Amount of memory to be allocated
* @tparam data_type The data type to be used for the buffer
* @tparam size_type The data type used for size calculations
*/
/* ----------------------------------------------------------------------------*/
template <typename data_type,
          typename size_type>
gdf_error allocSequenceBuffer(data_type ** buffer,
                         const size_type buffer_length) 
{
    RMM_TRY( RMM_ALLOC((void**)buffer, buffer_length*sizeof(data_type), 0) );
    thrust::sequence(thrust::device, *buffer, *buffer + buffer_length);
    return GDF_SUCCESS;
}

/* --------------------------------------------------------------------------*/
/** 
 * @Synopsis  Trivially computes full join of two tables if one of the tables
 are empty
 * 
 * @Param left_size The size of the left table
 * @Param right_size The size of the right table
 * @Param rightcol The right set of columns to join
 * @Param left_result The join computed indices of the left table
 * @Param right_result The join computed indices of the right table
 * @tparam size_type The data type used for size calculations
 * 
 * @Returns GDF_SUCCESS upon succesfull compute, otherwise returns appropriate error code
 */
/* ----------------------------------------------------------------------------*/
template<typename size_type>
gdf_error trivial_full_join(
        const size_type left_size,
        const size_type right_size,
        gdf_column *left_result,
        gdf_column *right_result) {
    // Deduce the type of the output gdf_columns
    gdf_dtype dtype;
    switch(sizeof(output_index_type))
    {
      case 1 : dtype = GDF_INT8;  break;
      case 2 : dtype = GDF_INT16; break;
      case 4 : dtype = GDF_INT32; break;
      case 8 : dtype = GDF_INT64; break;
    }

    output_index_type *l_ptr{nullptr};
    output_index_type *r_ptr{nullptr};
    size_type result_size{0};
    if ((left_size == 0) && (right_size == 0)) {
        return GDF_DATASET_EMPTY;
    }
    if (left_size == 0) {
        allocValueBuffer(&l_ptr, right_size,
                         static_cast<output_index_type>(-1));
        allocSequenceBuffer(&r_ptr, right_size);
        result_size = right_size;
    } else if (right_size == 0) {
        allocValueBuffer(&r_ptr, left_size,
                         static_cast<output_index_type>(-1));
        allocSequenceBuffer(&l_ptr, left_size);
        result_size = left_size;
    }
    gdf_column_view( left_result, l_ptr, nullptr, result_size, dtype);
    gdf_column_view(right_result, r_ptr, nullptr, result_size, dtype);
    CUDA_CHECK_LAST();
    return GDF_SUCCESS;
}

/* --------------------------------------------------------------------------*/
/** 
 * @Synopsis  Computes the join operation between two sets of columns
 * 
 * @Param num_cols The number of columns to join
 * @Param leftcol The left set of columns to join
 * @Param rightcol The right set of columns to join
 * @Param left_result The join computed indices of the left table
 * @Param right_result The join computed indices of the right table
 * @Param join_context A structure that determines various run parameters, such as
   whether to perform a hash or sort based join
 * @tparam join_type The type of join to be performed
 * 
 * @Returns GDF_SUCCESS upon succesfull compute, otherwise returns appropriate error code
 */
/* ----------------------------------------------------------------------------*/
template <JoinType join_type>
gdf_error join_call( int num_cols, gdf_column **leftcol, gdf_column **rightcol,
                     gdf_column *left_result, gdf_column *right_result,
                     gdf_context *join_context)
{

  using size_type = int64_t;

  if( (0 == num_cols) || (nullptr == leftcol) || (nullptr == rightcol))
    return GDF_DATASET_EMPTY;

  if(nullptr == join_context)
    return GDF_INVALID_API_CALL;

  const auto left_col_size = leftcol[0]->size;
  const auto right_col_size = rightcol[0]->size;
  
  // Check that the number of rows does not exceed the maximum
  if(left_col_size >= MAX_JOIN_SIZE) return GDF_COLUMN_SIZE_TOO_BIG;
  if(right_col_size >= MAX_JOIN_SIZE) return GDF_COLUMN_SIZE_TOO_BIG;


  // If both frames are empty, return immediately
  if((0 == left_col_size ) && (0 == right_col_size)) {
    return GDF_SUCCESS;
  }

  // If left join and the left table is empty, return immediately
  if( (JoinType::LEFT_JOIN == join_type) && (0 == left_col_size)){
    return GDF_SUCCESS;
  }

  // If Inner Join and either table is empty, return immediately
  if( (JoinType::INNER_JOIN == join_type) && 
      ((0 == left_col_size) || (0 == right_col_size)) ){
    return GDF_SUCCESS;
  }

  // If Inner Join and either table is empty, compute trivial full join
  if( (JoinType::FULL_JOIN == join_type) && 
      ((0 == left_col_size) || (0 == right_col_size)) ){
    return trivial_full_join<size_type>(left_col_size, right_col_size, left_result, right_result);
  }

  // check that the columns data are not null, have matching types, 
  // and the same number of rows
  for (int i = 0; i < num_cols; i++) {
    if((right_col_size > 0) && (nullptr == rightcol[i]->data)){
     return GDF_DATASET_EMPTY;
    } 
    if((left_col_size > 0) && (nullptr == leftcol[i]->data)){
     return GDF_DATASET_EMPTY;
    } 
    if(rightcol[i]->dtype != leftcol[i]->dtype) return GDF_DTYPE_MISMATCH;
    if(left_col_size != leftcol[i]->size) return GDF_COLUMN_SIZE_MISMATCH;
    if(right_col_size != rightcol[i]->size) return GDF_COLUMN_SIZE_MISMATCH;

    // Ensure GDF_TIMESTAMP columns have the same resolution
    if (GDF_TIMESTAMP == rightcol[i]->dtype) {
      GDF_REQUIRE(
          rightcol[i]->dtype_info.time_unit == leftcol[i]->dtype_info.time_unit,
          GDF_TIMESTAMP_RESOLUTION_MISMATCH);
    }
  }

  gdf_method join_method = join_context->flag_method; 

  gdf_error gdf_error_code{GDF_SUCCESS};

  PUSH_RANGE("LIBGDF_JOIN", JOIN_COLOR);

  switch(join_method)
  {
    case GDF_HASH:
      {
        gdf_error_code =  hash_join<join_type, size_type>(num_cols, leftcol, rightcol, left_result, right_result);
        break;
      }
    case GDF_SORT:
      {
        // Sort based joins only support single column joins
        if(1 == num_cols)
        {
          gdf_error_code =  sort_join<join_type>(leftcol[0], rightcol[0], left_result, right_result, join_context);
        }
        else
        {
          gdf_error_code =  GDF_JOIN_TOO_MANY_COLUMNS;
        }

        break;
      }
    default:
      gdf_error_code =  GDF_UNSUPPORTED_METHOD;
  }

  POP_RANGE();

  return gdf_error_code;
}

template <JoinType join_type, typename size_type, typename index_type>
gdf_error construct_join_output_df(
        std::vector<gdf_column*>& ljoincol,
        std::vector<gdf_column*>& rjoincol,
        gdf_column **left_cols, 
        int num_left_cols,
        int left_join_cols[],
        gdf_column **right_cols,
        int num_right_cols,
        int right_join_cols[],
        int num_cols_to_join,
        int result_num_cols,
        gdf_column ** result_cols,
        gdf_column * left_indices,
        gdf_column * right_indices) {

  PUSH_RANGE("LIBGDF_JOIN_OUTPUT", JOIN_COLOR);
    //create left and right input table with columns not joined on
    std::vector<gdf_column*> lnonjoincol;
    std::vector<gdf_column*> rnonjoincol;
    std::set<int> l_join_indices, r_join_indices;
    for (int i = 0; i < num_cols_to_join; ++i) {
        l_join_indices.insert(left_join_cols[i]);
        r_join_indices.insert(right_join_cols[i]);
    }
    for (int i = 0; i < num_left_cols; ++i) {
        if (l_join_indices.find(i) == l_join_indices.end()) {
            lnonjoincol.push_back(left_cols[i]);
        }
    }
    for (int i = 0; i < num_right_cols; ++i) {
        if (r_join_indices.find(i) == r_join_indices.end()) {
            rnonjoincol.push_back(right_cols[i]);
        }
    }
    //TODO : Invalid api

    size_type join_size = left_indices->size;
    int left_table_end = num_left_cols - num_cols_to_join;
    int right_table_begin = num_left_cols;

    //create left and right output column data buffers
    for (int i = 0; i < left_table_end; ++i) {
        gdf_column_view(result_cols[i], nullptr, nullptr, join_size, lnonjoincol[i]->dtype);
        int col_width; get_column_byte_width(result_cols[i], &col_width);
        RMM_TRY( RMM_ALLOC((void**)&(result_cols[i]->data), col_width * join_size, 0) ); // TODO: non-default stream?
        RMM_TRY( RMM_ALLOC((void**)&(result_cols[i]->valid), sizeof(gdf_valid_type)*gdf_get_num_chars_bitmask(join_size), 0) );
        CUDA_TRY( hipMemset(result_cols[i]->valid, 0, sizeof(gdf_valid_type)*gdf_get_num_chars_bitmask(join_size)) );
    }
    for (int i = right_table_begin; i < result_num_cols; ++i) {
        gdf_column_view(result_cols[i], nullptr, nullptr, join_size, rnonjoincol[i - right_table_begin]->dtype);
        int col_width; get_column_byte_width(result_cols[i], &col_width);
        RMM_TRY( RMM_ALLOC((void**)&(result_cols[i]->data), col_width * join_size, 0) ); // TODO: non-default stream?
        RMM_TRY( RMM_ALLOC((void**)&(result_cols[i]->valid), sizeof(gdf_valid_type)*gdf_get_num_chars_bitmask(join_size), 0) );
        CUDA_TRY( hipMemset(result_cols[i]->valid, 0, sizeof(gdf_valid_type)*gdf_get_num_chars_bitmask(join_size)) );
    }
    //create joined output column data buffers
    for (int join_index = 0; join_index < num_cols_to_join; ++join_index) {
        int i = left_table_end + join_index;
        gdf_column_view(result_cols[i], nullptr, nullptr, join_size, left_cols[left_join_cols[join_index]]->dtype);
        int col_width; get_column_byte_width(result_cols[i], &col_width);
        RMM_TRY( RMM_ALLOC((void**)&(result_cols[i]->data), col_width * join_size, 0) ); // TODO: non-default stream?
        RMM_TRY( RMM_ALLOC((void**)&(result_cols[i]->valid), sizeof(gdf_valid_type)*gdf_get_num_chars_bitmask(join_size), 0) );
        CUDA_TRY( hipMemset(result_cols[i]->valid, 0, sizeof(gdf_valid_type)*gdf_get_num_chars_bitmask(join_size)) );
    }

    gdf_error err{GDF_SUCCESS};

    //Construct the left columns
    if (0 != lnonjoincol.size()) {
        gdf_table<size_type> l_i_table(lnonjoincol.size(), lnonjoincol.data());
        gdf_table<size_type> l_table(num_left_cols - num_cols_to_join, result_cols);
        err = l_i_table.gather(static_cast<index_type*>(left_indices->data),
                l_table, join_type != JoinType::INNER_JOIN);
        if (err != GDF_SUCCESS) { return err; }
    }

    //Construct the right columns
    if (0 != rnonjoincol.size()) {
        gdf_table<size_type> r_i_table(rnonjoincol.size(), rnonjoincol.data());
        gdf_table<size_type> r_table(num_right_cols - num_cols_to_join, result_cols + right_table_begin);
        err = r_i_table.gather(static_cast<index_type*>(right_indices->data),
                r_table, join_type != JoinType::INNER_JOIN);
        if (err != GDF_SUCCESS) { return err; }
    }

    //Construct the joined columns
    if (0 != ljoincol.size()) {
        gdf_table<size_type> j_i_table(ljoincol.size(), ljoincol.data());
        gdf_table<size_type> j_table(num_cols_to_join, result_cols + left_table_end);
        //Gather valid rows from the right table
	// TODO: Revisit this, because it probably can be done more efficiently
        if (JoinType::FULL_JOIN == join_type) {
            gdf_table<size_type> j_i_r_table(rjoincol.size(), rjoincol.data());
            err = j_i_r_table.gather(static_cast<index_type*>(right_indices->data),
                    j_table, join_type != JoinType::INNER_JOIN);
            if (err != GDF_SUCCESS) { return err; }
        }
        err = j_i_table.gather(static_cast<index_type*>(left_indices->data),
                j_table, join_type != JoinType::INNER_JOIN);
    }

	POP_RANGE();
    return err;
}

template <JoinType join_type, typename size_type, typename index_type>
gdf_error join_call_compute_df(
                         gdf_column **left_cols, 
                         int num_left_cols,
                         int left_join_cols[],
                         gdf_column **right_cols,
                         int num_right_cols,
                         int right_join_cols[],
                         int num_cols_to_join,
                         int result_num_cols,
                         gdf_column **result_cols,
                         gdf_column * left_indices,
                         gdf_column * right_indices,
                         gdf_context *join_context) {
  GDF_REQUIRE(nullptr != left_cols, GDF_DATASET_EMPTY);
  GDF_REQUIRE(nullptr != right_cols, GDF_DATASET_EMPTY);

  // If the inputs are empty, immediately return
  if( (0 == left_cols[0]->size) && (0 == right_cols[0]->size) ){
    return GDF_SUCCESS;
  }

  GDF_REQUIRE(0 != num_cols_to_join, GDF_SUCCESS);

  GDF_REQUIRE(nullptr != left_join_cols, GDF_DATASET_EMPTY);
  GDF_REQUIRE(nullptr != right_join_cols, GDF_DATASET_EMPTY);
  GDF_REQUIRE(nullptr != join_context, GDF_INVALID_API_CALL);

  // Determine if requested output is the indices of matching rows, the fully
  // constructed output dataframe, or both
  bool const construct_output_dataframe{nullptr != result_cols};
  bool const return_output_indices{(nullptr != left_indices) and
                                   (nullptr != right_indices)};

  GDF_REQUIRE(construct_output_dataframe or return_output_indices,
              GDF_INVALID_API_CALL);

  // If index outputs are not requested, create columns to store them
  // for computing combined join output
  gdf_column *left_index_out = left_indices;
  gdf_column *right_index_out = right_indices;

  using gdf_col_pointer =
      typename std::unique_ptr<gdf_column, std::function<void(gdf_column *)>>;
  auto gdf_col_deleter = [](gdf_column *col) {
    col->size = 0;
    if (col->data) {
      RMM_FREE(col->data, 0);
    }
    if (col->valid) {
      RMM_FREE(col->valid, 0);
    }
  };
  gdf_col_pointer l_index_temp, r_index_temp;

  if (nullptr == left_indices) {
    l_index_temp = {new gdf_column, gdf_col_deleter};
    left_index_out = l_index_temp.get();
    }

    if (nullptr == right_indices) {
        r_index_temp = {new gdf_column, gdf_col_deleter};
        right_index_out = r_index_temp.get();
    }

    //get column pointers to join on
    std::vector<gdf_column*> ljoincol;
    std::vector<gdf_column*> rjoincol;
    for (int i = 0; i < num_cols_to_join; ++i) {
        ljoincol.push_back(left_cols[ left_join_cols[i] ]);
        rjoincol.push_back(right_cols[ right_join_cols[i] ]);
    }


    gdf_error join_err = join_call<join_type>(num_cols_to_join,
            ljoincol.data(), rjoincol.data(),
            left_index_out, right_index_out,
            join_context);
    //If construct_output_dataframe is false then left_index_out or right_index_out
    //was not dynamically allocated.
    if ((!construct_output_dataframe) || (GDF_SUCCESS != join_err)) {
        return join_err;
    }

    gdf_error df_err =
        construct_join_output_df<join_type, size_type, index_type>(
            ljoincol, rjoincol,
            left_cols, num_left_cols, left_join_cols,
            right_cols, num_right_cols, right_join_cols,
            num_cols_to_join, result_num_cols, result_cols,
            left_index_out, right_index_out);

    l_index_temp.reset(nullptr);
    r_index_temp.reset(nullptr);

    CUDA_CHECK_LAST();

    return df_err;
}

gdf_error gdf_left_join(
                         gdf_column **left_cols, 
                         int num_left_cols,
                         int left_join_cols[],
                         gdf_column **right_cols,
                         int num_right_cols,
                         int right_join_cols[],
                         int num_cols_to_join,
                         int result_num_cols,
                         gdf_column **result_cols,
                         gdf_column * left_indices,
                         gdf_column * right_indices,
                         gdf_context *join_context) {
    return join_call_compute_df<JoinType::LEFT_JOIN, int64_t, output_index_type>(
                     left_cols, 
                     num_left_cols,
                     left_join_cols,
                     right_cols,
                     num_right_cols,
                     right_join_cols,
                     num_cols_to_join,
                     result_num_cols,
                     result_cols,
                     left_indices,
                     right_indices,
                     join_context);
}

gdf_error gdf_inner_join(
                         gdf_column **left_cols, 
                         int num_left_cols,
                         int left_join_cols[],
                         gdf_column **right_cols,
                         int num_right_cols,
                         int right_join_cols[],
                         int num_cols_to_join,
                         int result_num_cols,
                         gdf_column **result_cols,
                         gdf_column * left_indices,
                         gdf_column * right_indices,
                         gdf_context *join_context) {
    return join_call_compute_df<JoinType::INNER_JOIN, int64_t, output_index_type>(
                     left_cols, 
                     num_left_cols,
                     left_join_cols,
                     right_cols,
                     num_right_cols,
                     right_join_cols,
                     num_cols_to_join,
                     result_num_cols,
                     result_cols,
                     left_indices,
                     right_indices,
                     join_context);
}

gdf_error gdf_full_join(
                         gdf_column **left_cols, 
                         int num_left_cols,
                         int left_join_cols[],
                         gdf_column **right_cols,
                         int num_right_cols,
                         int right_join_cols[],
                         int num_cols_to_join,
                         int result_num_cols,
                         gdf_column **result_cols,
                         gdf_column * left_indices,
                         gdf_column * right_indices,
                         gdf_context *join_context) {
    return join_call_compute_df<JoinType::FULL_JOIN, int64_t, output_index_type>(
                     left_cols, 
                     num_left_cols,
                     left_join_cols,
                     right_cols,
                     num_right_cols,
                     right_join_cols,
                     num_cols_to_join,
                     result_num_cols,
                     result_cols,
                     left_indices,
                     right_indices,
                     join_context);
}
