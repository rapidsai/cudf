#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2017, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */


#include <limits>
#include <set>
#include <vector>

#include "cudf.h"
#include "rmm/rmm.h"
#include "utilities/error_utils.h"
#include "dataframe/cudf_table.cuh"
#include "utilities/nvtx/nvtx_utils.h"

#include "join_types.h"
#include "joining.h"

// Size limit due to use of int32 as join output.
// FIXME: upgrade to 64-bit
using output_index_type = int;
constexpr output_index_type MAX_JOIN_SIZE{std::numeric_limits<output_index_type>::max()};

/* --------------------------------------------------------------------------*/
/** 
 * @Synopsis Computes the Join result between two tables using the hash-based implementation. 
 * 
 * @Param num_cols The number of columns to join
 * @Param leftcol The left set of columns to join
 * @Param rightcol The right set of columns to join
 * @Param l_result The join computed indices of the left table
 * @Param r_result The join computed indices of the right table
 * @tparam join_type The type of join to be performed
 * @tparam size_type The data type used for size calculations
 * 
 * @Returns Upon successful computation, returns GDF_SUCCESS. Otherwise returns appropriate error code 
 */
/* ----------------------------------------------------------------------------*/
template <JoinType join_type, 
          typename size_type>
gdf_error hash_join(size_type num_cols, gdf_column **leftcol, gdf_column **rightcol,
                    gdf_column *l_result, gdf_column *r_result)
{
  // Wrap the set of gdf_columns in a gdf_table class
  std::unique_ptr< gdf_table<size_type> > left_table(new gdf_table<size_type>(num_cols, leftcol));
  std::unique_ptr< gdf_table<size_type> > right_table(new gdf_table<size_type>(num_cols, rightcol));

  return join_hash<join_type, output_index_type>(*left_table, 
                                                        *right_table, 
                                                        l_result, 
                                                        r_result);
}

/* --------------------------------------------------------------------------*/
/** 
 * @Synopsis Computes the Join result between two tables using the sort-based implementation. 
 * 
 * @Param num_cols The number of columns to join
 * @Param leftcol The left set of columns to join
 * @Param rightcol The right set of columns to join
 * @Param out_result The result of the join operation. The first n/2 elements of the
   output are the left indices, the last n/2 elements of the output are the right indices.
   @tparam join_type The type of join to be performed
 * 
 * @Returns Upon successful computation, returns GDF_SUCCESS. Otherwise returns appropriate error code 
 */
/* ----------------------------------------------------------------------------*/
template <JoinType join_type, 
          typename size_type>
gdf_error sort_join(size_type num_cols, gdf_column **leftcol, gdf_column **rightcol,
                    gdf_column *l_result, gdf_column *r_result)
{
  if (num_cols > 1) {
    return GDF_JOIN_TOO_MANY_COLUMNS;
  } else if (num_cols == 0) {
    return GDF_DATASET_EMPTY;
  } else if ((leftcol[0]->null_count != 0) || (rightcol[0]->null_count != 0)) {
      return GDF_VALIDITY_UNSUPPORTED;
  }

  return join_sort<join_type, output_index_type>(
          leftcol[0], rightcol[0], l_result, r_result);
}

/* --------------------------------------------------------------------------*/
/**
* @Synopsis  Allocates a buffer and fills it with a repeated value
*
* @Param buffer Address of the buffer to be allocated
* @Param buffer_length Amount of memory to be allocated
* @Param value The value to be filled into the buffer
* @tparam data_type The data type to be used for the buffer
* @tparam size_type The data type used for size calculations
*/
/* ----------------------------------------------------------------------------*/
template <typename data_type,
          typename size_type>
gdf_error allocValueBuffer(data_type ** buffer,
                           const size_type buffer_length,
                           const data_type value) 
{
    RMM_TRY( RMM_ALLOC((void**)buffer, buffer_length*sizeof(data_type), 0) );
    thrust::fill(thrust::device, *buffer, *buffer + buffer_length, value);
    return GDF_SUCCESS;
}

/* --------------------------------------------------------------------------*/
/**
* @Synopsis  Allocates a buffer and fills it with a sequence
*
* @Param buffer Address of the buffer to be allocated
* @Param buffer_length Amount of memory to be allocated
* @tparam data_type The data type to be used for the buffer
* @tparam size_type The data type used for size calculations
*/
/* ----------------------------------------------------------------------------*/
template <typename data_type,
          typename size_type>
gdf_error allocSequenceBuffer(data_type ** buffer,
                         const size_type buffer_length) 
{
    RMM_TRY( RMM_ALLOC((void**)buffer, buffer_length*sizeof(data_type), 0) );
    thrust::sequence(thrust::device, *buffer, *buffer + buffer_length);
    return GDF_SUCCESS;
}

/* --------------------------------------------------------------------------*/
/** 
 * @Synopsis  Trivially computes full join of two tables if one of the tables
 are empty
 * 
 * @Param left_size The size of the left table
 * @Param right_size The size of the right table
 * @Param rightcol The right set of columns to join
 * @Param left_result The join computed indices of the left table
 * @Param right_result The join computed indices of the right table
 * @tparam size_type The data type used for size calculations
 * 
 * @Returns GDF_SUCCESS upon succesfull compute, otherwise returns appropriate error code
 */
/* ----------------------------------------------------------------------------*/
template<typename size_type>
gdf_error trivial_full_join(
        const size_type left_size,
        const size_type right_size,
        gdf_column *left_result,
        gdf_column *right_result) {
    // Deduce the type of the output gdf_columns
    gdf_dtype dtype;
    switch(sizeof(output_index_type))
    {
      case 1 : dtype = GDF_INT8;  break;
      case 2 : dtype = GDF_INT16; break;
      case 4 : dtype = GDF_INT32; break;
      case 8 : dtype = GDF_INT64; break;
    }

    output_index_type *l_ptr{nullptr};
    output_index_type *r_ptr{nullptr};
    size_type result_size{0};
    if ((left_size == 0) && (right_size == 0)) {
        return GDF_DATASET_EMPTY;
    }
    if (left_size == 0) {
        allocValueBuffer(&l_ptr, right_size,
                         static_cast<output_index_type>(-1));
        allocSequenceBuffer(&r_ptr, right_size);
        result_size = right_size;
    } else if (right_size == 0) {
        allocValueBuffer(&r_ptr, left_size,
                         static_cast<output_index_type>(-1));
        allocSequenceBuffer(&l_ptr, left_size);
        result_size = left_size;
    }
    gdf_column_view( left_result, l_ptr, nullptr, result_size, dtype);
    gdf_column_view(right_result, r_ptr, nullptr, result_size, dtype);
    CUDA_CHECK_LAST();
    return GDF_SUCCESS;
}

/* --------------------------------------------------------------------------*/
/** 
 * @Synopsis  Computes the join operation between two sets of columns
 * 
 * @Param num_cols The number of columns to join
 * @Param leftcol The left set of columns to join
 * @Param rightcol The right set of columns to join
 * @Param left_result The join computed indices of the left table
 * @Param right_result The join computed indices of the right table
 * @Param join_context A structure that determines various run parameters, such as
   whether to perform a hash or sort based join
 * @tparam join_type The type of join to be performed
 * 
 * @Returns GDF_SUCCESS upon succesfull compute, otherwise returns appropriate error code
 */
/* ----------------------------------------------------------------------------*/
template <JoinType join_type>
gdf_error join_call( int num_cols, gdf_column **leftcol, gdf_column **rightcol,
                     gdf_column *left_result, gdf_column *right_result,
                     gdf_context *join_context)
{

  using size_type = int64_t;

  if( (0 == num_cols) || (nullptr == leftcol) || (nullptr == rightcol))
    return GDF_DATASET_EMPTY;

  if(nullptr == join_context)
    return GDF_INVALID_API_CALL;

  const auto left_col_size = leftcol[0]->size;
  const auto right_col_size = rightcol[0]->size;
  
  // Check that the number of rows does not exceed the maximum
  if(left_col_size >= MAX_JOIN_SIZE) return GDF_COLUMN_SIZE_TOO_BIG;
  if(right_col_size >= MAX_JOIN_SIZE) return GDF_COLUMN_SIZE_TOO_BIG;

  // If both frames are empty, return immediately
  if((0 == left_col_size ) && (0 == right_col_size)) {
    return GDF_SUCCESS;
  }

  // If left join and the left table is empty, return immediately
  if( (JoinType::LEFT_JOIN == join_type) && (0 == left_col_size)){
    return GDF_SUCCESS;
  }

  // If Inner Join and either table is empty, return immediately
  if( (JoinType::INNER_JOIN == join_type) && 
      ((0 == left_col_size) || (0 == right_col_size)) ){
    return GDF_SUCCESS;
  }

  // If Inner Join and either table is empty, compute trivial full join
  if( (JoinType::FULL_JOIN == join_type) && 
      ((0 == left_col_size) || (0 == right_col_size)) ){
    return trivial_full_join<size_type>(left_col_size, right_col_size, left_result, right_result);
  }

  // check that the columns data are not null, have matching types, 
  // and the same number of rows
  for (int i = 0; i < num_cols; i++) {
    if((right_col_size > 0) && (nullptr == rightcol[i]->data)){
     return GDF_DATASET_EMPTY;
    } 
    if((left_col_size > 0) && (nullptr == leftcol[i]->data)){
     return GDF_DATASET_EMPTY;
    } 
    if(rightcol[i]->dtype != leftcol[i]->dtype) return GDF_JOIN_DTYPE_MISMATCH;
    if(left_col_size != leftcol[i]->size) return GDF_COLUMN_SIZE_MISMATCH;
    if(right_col_size != rightcol[i]->size) return GDF_COLUMN_SIZE_MISMATCH;
  }

  gdf_method join_method = join_context->flag_method; 

  gdf_error gdf_error_code{GDF_SUCCESS};

  PUSH_RANGE("LIBGDF_JOIN", JOIN_COLOR);

  switch(join_method)
  {
    case GDF_HASH:
      {
        gdf_error_code =  hash_join<join_type, size_type>(num_cols, leftcol, rightcol, left_result, right_result);
        break;
      }
    case GDF_SORT:
      {
        gdf_error_code =  sort_join<join_type, size_type>(num_cols, leftcol, rightcol, left_result, right_result);
        break;
      }
    default:
      gdf_error_code =  GDF_UNSUPPORTED_METHOD;
  }

  POP_RANGE();

  return gdf_error_code;
}

template <JoinType join_type, typename size_type, typename index_type>
gdf_error construct_join_output_df(
        std::vector<gdf_column*>& ljoincol,
        std::vector<gdf_column*>& rjoincol,
        gdf_column **left_cols, 
        int num_left_cols,
        int left_join_cols[],
        gdf_column **right_cols,
        int num_right_cols,
        int right_join_cols[],
        int num_cols_to_join,
        int result_num_cols,
        gdf_column ** result_cols,
        gdf_column * left_indices,
        gdf_column * right_indices) {

  PUSH_RANGE("LIBGDF_JOIN_OUTPUT", JOIN_COLOR);
    //create left and right input table with columns not joined on
    std::vector<gdf_column*> lnonjoincol;
    std::vector<gdf_column*> rnonjoincol;
    std::set<int> l_join_indices, r_join_indices;
    for (int i = 0; i < num_cols_to_join; ++i) {
        l_join_indices.insert(left_join_cols[i]);
        r_join_indices.insert(right_join_cols[i]);
    }
    for (int i = 0; i < num_left_cols; ++i) {
        if (l_join_indices.find(i) == l_join_indices.end()) {
            lnonjoincol.push_back(left_cols[i]);
        }
    }
    for (int i = 0; i < num_right_cols; ++i) {
        if (r_join_indices.find(i) == r_join_indices.end()) {
            rnonjoincol.push_back(right_cols[i]);
        }
    }
    //TODO : Invalid api

    size_type join_size = left_indices->size;
    int left_table_end = num_left_cols - num_cols_to_join;
    int right_table_begin = num_left_cols;

    //create left and right output column data buffers
    for (int i = 0; i < left_table_end; ++i) {
        gdf_column * input_column = lnonjoincol[i];
        gdf_column_view(result_cols[i], nullptr, nullptr, join_size, input_column->dtype);
        int col_width; get_column_byte_width(result_cols[i], &col_width);
        RMM_TRY( RMM_ALLOC((void**)&(result_cols[i]->data), col_width * join_size, 0) ); // TODO: non-default stream?
        RMM_TRY( RMM_ALLOC((void**)&(result_cols[i]->valid), sizeof(gdf_valid_type)*gdf_get_num_chars_bitmask(join_size), 0) );
        CUDA_TRY( hipMemset(result_cols[i]->valid, 0xff, sizeof(gdf_valid_type)*gdf_get_num_chars_bitmask(join_size)) );
        result_cols[i]->null_count = 0;
    }
    for (int i = right_table_begin; i < result_num_cols; ++i) {
        gdf_column * input_column = rnonjoincol[i - right_table_begin];
        gdf_column_view(result_cols[i], nullptr, nullptr, join_size, input_column->dtype);
        int col_width; get_column_byte_width(result_cols[i], &col_width);
        RMM_TRY( RMM_ALLOC((void**)&(result_cols[i]->data), col_width * join_size, 0) ); // TODO: non-default stream?
        RMM_TRY( RMM_ALLOC((void**)&(result_cols[i]->valid), sizeof(gdf_valid_type)*gdf_get_num_chars_bitmask(join_size), 0) );
        CUDA_TRY( hipMemset(result_cols[i]->valid, 0xff, sizeof(gdf_valid_type)*gdf_get_num_chars_bitmask(join_size)) );
        result_cols[i]->null_count = 0;
    }
    //create joined output column data buffers
    for (int join_index = 0; join_index < num_cols_to_join; ++join_index) {
        int i = left_table_end + join_index;
        gdf_column * input_column = left_cols[left_join_cols[join_index]];
        gdf_column_view(result_cols[i], nullptr, nullptr, join_size, input_column->dtype);
        int col_width; get_column_byte_width(result_cols[i], &col_width);
        RMM_TRY( RMM_ALLOC((void**)&(result_cols[i]->data), col_width * join_size, 0) ); // TODO: non-default stream?
        RMM_TRY( RMM_ALLOC((void**)&(result_cols[i]->valid), sizeof(gdf_valid_type)*gdf_get_num_chars_bitmask(join_size), 0) );
        CUDA_TRY( hipMemset(result_cols[i]->valid, 0xff, sizeof(gdf_valid_type)*gdf_get_num_chars_bitmask(join_size)) );
        result_cols[i]->null_count = 0;
    }

    gdf_error err{GDF_SUCCESS};

    //Construct the left columns
    if (0 != lnonjoincol.size()) {
        gdf_table<size_type> l_i_table(lnonjoincol.size(), lnonjoincol.data());
        gdf_table<size_type> l_table(num_left_cols - num_cols_to_join, result_cols);
        err = l_i_table.gather(static_cast<index_type*>(left_indices->data),
                l_table, join_type != JoinType::INNER_JOIN);
        if (err != GDF_SUCCESS) { return err; }
    }

    //Construct the right columns
    if (0 != rnonjoincol.size()) {
        gdf_table<size_type> r_i_table(rnonjoincol.size(), rnonjoincol.data());
        gdf_table<size_type> r_table(num_right_cols - num_cols_to_join, result_cols + right_table_begin);
        err = r_i_table.gather(static_cast<index_type*>(right_indices->data),
                r_table, join_type != JoinType::INNER_JOIN);
        if (err != GDF_SUCCESS) { return err; }
    }

    //Construct the joined columns
    if (0 != ljoincol.size()) {
        gdf_table<size_type> j_i_table(ljoincol.size(), ljoincol.data());
        gdf_table<size_type> j_table(num_cols_to_join, result_cols + left_table_end);
        //Gather valid rows from the right table
	// TODO: Revisit this, because it probably can be done more efficiently
        if (JoinType::FULL_JOIN == join_type) {
            gdf_table<size_type> j_i_r_table(rjoincol.size(), rjoincol.data());
            err = j_i_r_table.gather(static_cast<index_type*>(right_indices->data),
                    j_table, join_type != JoinType::INNER_JOIN);
            if (err != GDF_SUCCESS) { return err; }
        }
        err = j_i_table.gather(static_cast<index_type*>(left_indices->data),
                j_table, join_type != JoinType::INNER_JOIN);
    }

	POP_RANGE();
    return err;
}

template <JoinType join_type, typename size_type, typename index_type>
gdf_error join_call_compute_df(
                         gdf_column **left_cols, 
                         int num_left_cols,
                         int left_join_cols[],
                         gdf_column **right_cols,
                         int num_right_cols,
                         int right_join_cols[],
                         int num_cols_to_join,
                         int result_num_cols,
                         gdf_column **result_cols,
                         gdf_column * left_indices,
                         gdf_column * right_indices,
                         gdf_context *join_context) {
    //return error if the inputs are invalid
    if ((left_cols == nullptr)  ||
        (right_cols == nullptr)) { return GDF_DATASET_EMPTY; }

    if (num_cols_to_join == 0) { return GDF_SUCCESS; }
    
    if ((left_join_cols == nullptr)  ||
        (right_join_cols == nullptr)) { return GDF_DATASET_EMPTY; }

    //check if combined join output is expected
    bool compute_df = (result_cols != nullptr);

    //return error if no output pointers are valid
    if ( ((left_indices == nullptr)||(right_indices == nullptr)) &&
         (!compute_df) ) { return GDF_DATASET_EMPTY; }

    if (join_context == nullptr) { return GDF_INVALID_API_CALL; }

    //If index outputs are not requested, create columns to store them
    //for computing combined join output
    gdf_column * left_index_out = left_indices;
    gdf_column * right_index_out = right_indices;

    using gdf_col_pointer = typename std::unique_ptr<gdf_column, std::function<void(gdf_column*)>>;
    auto gdf_col_deleter = [](gdf_column* col){
        col->size = 0;
        if (col->data)  { RMM_FREE(col->data, 0);  }
        if (col->valid) { RMM_FREE(col->valid, 0); }
    };
    gdf_col_pointer l_index_temp, r_index_temp;

    if (nullptr == left_indices) {
        l_index_temp = {new gdf_column, gdf_col_deleter};
        left_index_out = l_index_temp.get();
    }

    if (nullptr == right_indices) {
        r_index_temp = {new gdf_column, gdf_col_deleter};
        right_index_out = r_index_temp.get();
    }

    //get column pointers to join on
    std::vector<gdf_column*> ljoincol;
    std::vector<gdf_column*> rjoincol;
    for (int i = 0; i < num_cols_to_join; ++i) {
        ljoincol.push_back(left_cols[ left_join_cols[i] ]);
        rjoincol.push_back(right_cols[ right_join_cols[i] ]);
    }


    gdf_error join_err = join_call<join_type>(num_cols_to_join,
            ljoincol.data(), rjoincol.data(),
            left_index_out, right_index_out,
            join_context);
    //If compute_df is false then left_index_out or right_index_out
    //was not dynamically allocated.
    if ((!compute_df) || (GDF_SUCCESS != join_err)) {
        return join_err;
    }

    gdf_error df_err =
        construct_join_output_df<join_type, size_type, index_type>(
            ljoincol, rjoincol,
            left_cols, num_left_cols, left_join_cols,
            right_cols, num_right_cols, right_join_cols,
            num_cols_to_join, result_num_cols, result_cols,
            left_index_out, right_index_out);

    l_index_temp.reset(nullptr);
    r_index_temp.reset(nullptr);

    CUDA_CHECK_LAST();

    return df_err;
}

gdf_error gdf_left_join(
                         gdf_column **left_cols, 
                         int num_left_cols,
                         int left_join_cols[],
                         gdf_column **right_cols,
                         int num_right_cols,
                         int right_join_cols[],
                         int num_cols_to_join,
                         int result_num_cols,
                         gdf_column **result_cols,
                         gdf_column * left_indices,
                         gdf_column * right_indices,
                         gdf_context *join_context) {
    return join_call_compute_df<JoinType::LEFT_JOIN, int64_t, output_index_type>(
                     left_cols, 
                     num_left_cols,
                     left_join_cols,
                     right_cols,
                     num_right_cols,
                     right_join_cols,
                     num_cols_to_join,
                     result_num_cols,
                     result_cols,
                     left_indices,
                     right_indices,
                     join_context);
}

gdf_error gdf_inner_join(
                         gdf_column **left_cols, 
                         int num_left_cols,
                         int left_join_cols[],
                         gdf_column **right_cols,
                         int num_right_cols,
                         int right_join_cols[],
                         int num_cols_to_join,
                         int result_num_cols,
                         gdf_column **result_cols,
                         gdf_column * left_indices,
                         gdf_column * right_indices,
                         gdf_context *join_context) {
    return join_call_compute_df<JoinType::INNER_JOIN, int64_t, output_index_type>(
                     left_cols, 
                     num_left_cols,
                     left_join_cols,
                     right_cols,
                     num_right_cols,
                     right_join_cols,
                     num_cols_to_join,
                     result_num_cols,
                     result_cols,
                     left_indices,
                     right_indices,
                     join_context);
}

gdf_error gdf_full_join(
                         gdf_column **left_cols, 
                         int num_left_cols,
                         int left_join_cols[],
                         gdf_column **right_cols,
                         int num_right_cols,
                         int right_join_cols[],
                         int num_cols_to_join,
                         int result_num_cols,
                         gdf_column **result_cols,
                         gdf_column * left_indices,
                         gdf_column * right_indices,
                         gdf_context *join_context) {
    return join_call_compute_df<JoinType::FULL_JOIN, int64_t, output_index_type>(
                     left_cols, 
                     num_left_cols,
                     left_join_cols,
                     right_cols,
                     num_right_cols,
                     right_join_cols,
                     num_cols_to_join,
                     result_num_cols,
                     result_cols,
                     left_indices,
                     right_indices,
                     join_context);
}
