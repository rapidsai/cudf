#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2017, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */



#include <cudf/types.hpp>
#include <cudf/cudf.h>
#include <rmm/rmm.h>
#include <utilities/column_utils.hpp>
#include <utilities/error_utils.hpp>
#include <cudf/utilities/legacy/type_dispatcher.hpp>
#include <utilities/nvtx/nvtx_utils.h>
#include <cudf/utilities/legacy/nvcategory_util.hpp>
#include <nvstrings/NVCategory.h>
#include <copying/gather.hpp>
#include "joining.h"

#include <limits>
#include <set>
#include <vector>

// Size limit due to use of int32 as join output.
// FIXME: upgrade to 64-bit
using output_index_type = gdf_index_type;
constexpr output_index_type MAX_JOIN_SIZE{std::numeric_limits<output_index_type>::max()};

/* --------------------------------------------------------------------------*/
/** 
 * @brief Computes the Join result between two tables using the hash-based implementation. 
 * 
 * @param[in] num_cols The number of columns to join
 * @param[in] leftcol The left set of columns to join
 * @param[in] rightcol The right set of columns to join
 * @param[out] l_result The join computed indices of the left table
 * @param[out] r_result The join computed indices of the right table
 * @tparam join_type The type of join to be performed
 * 
 * @returns Upon successful computation, returns GDF_SUCCESS. Otherwise returns appropriate error code 
 */
/* ----------------------------------------------------------------------------*/
template <JoinType join_type>
gdf_error hash_join(gdf_size_type num_cols, gdf_column **leftcol, gdf_column **rightcol,
                    gdf_column *l_result, gdf_column *r_result)
{
  cudf::table left_table{leftcol, num_cols};
  cudf::table right_table{rightcol, num_cols};

  return join_hash<join_type, output_index_type>(left_table, right_table,
                                                 l_result, r_result);
}

/* --------------------------------------------------------------------------*/
/**
 * @brief  Allocates a buffer and fills it with a repeated value
 *
 * @param[in,out] buffer Address of the buffer to be allocated
 * @param[in] buffer_length Amount of memory to be allocated
 * @param[in] value The value to be filled into the buffer
 * @tparam data_type The data type to be used for the buffer
 * 
 * @returns GDF_SUCCESS upon succesful completion
 */
/* ----------------------------------------------------------------------------*/
template <typename data_type>
gdf_error allocValueBuffer(data_type ** buffer,
                           const gdf_size_type buffer_length,
                           const data_type value) 
{
    RMM_TRY( RMM_ALLOC((void**)buffer, buffer_length*sizeof(data_type), 0) );
    thrust::fill(thrust::device, *buffer, *buffer + buffer_length, value);
    return GDF_SUCCESS;
}

/* --------------------------------------------------------------------------*/
/**
 * @brief  Allocates a buffer and fills it with a sequence
 *
 * @param[in,out] buffer Address of the buffer to be allocated
 * @param[in] buffer_length Amount of memory to be allocated
 * @tparam data_type The data type to be used for the buffer
 * 
 * @returns GDF_SUCCESS upon succesful completion
 */
/* ----------------------------------------------------------------------------*/
template <typename data_type>
gdf_error allocSequenceBuffer(data_type ** buffer,
                              const gdf_size_type buffer_length) 
{
    RMM_TRY( RMM_ALLOC((void**)buffer, buffer_length*sizeof(data_type), 0) );
    thrust::sequence(thrust::device, *buffer, *buffer + buffer_length);
    return GDF_SUCCESS;
}

/* --------------------------------------------------------------------------*/
/** 
 * @brief  Trivially computes full join of two tables if one of the tables
 * are empty
 * 
 * @param[in] left_size The size of the left table
 * @param[in] right_size The size of the right table
 * @param[in] rightcol The right set of columns to join
 * @param[out] left_result The join computed indices of the left table
 * @param[out] right_result The join computed indices of the right table
 * 
 * @returns GDF_SUCCESS upon succesfull compute, otherwise returns appropriate error code
 */
/* ----------------------------------------------------------------------------*/
gdf_error trivial_full_join(
        const gdf_size_type left_size,
        const gdf_size_type right_size,
        gdf_column *left_result,
        gdf_column *right_result) {
    // Deduce the type of the output gdf_columns
    gdf_dtype dtype;
    switch(sizeof(output_index_type))
    {
      case 1 : dtype = GDF_INT8;  break;
      case 2 : dtype = GDF_INT16; break;
      case 4 : dtype = GDF_INT32; break;
      case 8 : dtype = GDF_INT64; break;
    }

    output_index_type *l_ptr{nullptr};
    output_index_type *r_ptr{nullptr};
    gdf_size_type result_size{0};
    if ((left_size == 0) && (right_size == 0)) {
        return GDF_DATASET_EMPTY;
    }
    if (left_size == 0) {
        allocValueBuffer(&l_ptr, right_size,
                         static_cast<output_index_type>(-1));
        allocSequenceBuffer(&r_ptr, right_size);
        result_size = right_size;
    } else if (right_size == 0) {
        allocValueBuffer(&r_ptr, left_size,
                         static_cast<output_index_type>(-1));
        allocSequenceBuffer(&l_ptr, left_size);
        result_size = left_size;
    }

    gdf_column_view_augmented(left_result,
                              l_ptr, nullptr,
                              result_size, dtype, 0,
                              left_result->dtype_info,
                              left_result->col_name);

    gdf_column_view_augmented(right_result,
                              r_ptr, nullptr,
                              result_size, dtype, 0,
                              right_result->dtype_info,
                              right_result->col_name);

    CUDA_CHECK_LAST();
    return GDF_SUCCESS;
}

/* --------------------------------------------------------------------------*/
/** 
 * @brief  Computes the join operation between two sets of columns
 * 
 * @param[in] num_cols The number of columns to join
 * @param[in] leftcol The left set of columns to join
 * @param[in] rightcol The right set of columns to join
 * @param[out] left_result The join computed indices of the left table
 * @param[out] right_result The join computed indices of the right table
 * @param[in] join_context A structure that determines various run parameters, such as
 *                         whether to perform a hash or sort based join
 * @tparam join_type The type of join to be performed
 * 
 * @returns GDF_SUCCESS upon succesfull compute, otherwise returns appropriate error code
 */
/* ----------------------------------------------------------------------------*/
template <JoinType join_type>
gdf_error join_call( int num_cols, gdf_column **leftcol, gdf_column **rightcol,
                     gdf_column *left_result, gdf_column *right_result,
                     gdf_context *join_context)
{
  GDF_REQUIRE( 0 != num_cols, GDF_DATASET_EMPTY);
  GDF_REQUIRE( nullptr != leftcol, GDF_DATASET_EMPTY);
  GDF_REQUIRE( nullptr != rightcol, GDF_DATASET_EMPTY);
  GDF_REQUIRE( nullptr != join_context, GDF_INVALID_API_CALL);

  const auto left_col_size = leftcol[0]->size;
  const auto right_col_size = rightcol[0]->size;
  
  GDF_REQUIRE( left_col_size < MAX_JOIN_SIZE, GDF_COLUMN_SIZE_TOO_BIG);
  GDF_REQUIRE( right_col_size < MAX_JOIN_SIZE, GDF_COLUMN_SIZE_TOO_BIG);


  // If both frames are empty, return immediately
  if((0 == left_col_size ) && (0 == right_col_size)) {
    return GDF_SUCCESS;
  }

  // If left join and the left table is empty, return immediately
  if( (JoinType::LEFT_JOIN == join_type) && (0 == left_col_size)){
    return GDF_SUCCESS;
  }

  // If Inner Join and either table is empty, return immediately
  if( (JoinType::INNER_JOIN == join_type) && 
      ((0 == left_col_size) || (0 == right_col_size)) ){
    return GDF_SUCCESS;
  }

  // If Full Join and either table is empty, compute trivial full join
  if( (JoinType::FULL_JOIN == join_type) && 
      ((0 == left_col_size) || (0 == right_col_size)) ){
    return trivial_full_join(left_col_size, right_col_size, left_result, right_result);
  }

  // check that the columns data are not null, have matching types, 
  // and the same number of rows
  for (int i = 0; i < num_cols; i++) {
    if((right_col_size > 0) && (nullptr == rightcol[i]->data)){
     return GDF_DATASET_EMPTY;
    } 
    if((left_col_size > 0) && (nullptr == leftcol[i]->data)){
     return GDF_DATASET_EMPTY;
    } 
    if(rightcol[i]->dtype != leftcol[i]->dtype) return GDF_DTYPE_MISMATCH;
    if(left_col_size != leftcol[i]->size) return GDF_COLUMN_SIZE_MISMATCH;
    if(right_col_size != rightcol[i]->size) return GDF_COLUMN_SIZE_MISMATCH;

    // Ensure GDF_TIMESTAMP columns have the same resolution
    if (GDF_TIMESTAMP == rightcol[i]->dtype) {
      GDF_REQUIRE(
          rightcol[i]->dtype_info.time_unit == leftcol[i]->dtype_info.time_unit,
          GDF_TIMESTAMP_RESOLUTION_MISMATCH);
    }
  }

  gdf_method join_method = join_context->flag_method; 

  gdf_error gdf_error_code{GDF_SUCCESS};

  PUSH_RANGE("LIBGDF_JOIN", JOIN_COLOR);

  switch(join_method)
  {
    case GDF_HASH:
      {
        gdf_error_code =  hash_join<join_type>(num_cols, leftcol, rightcol, left_result, right_result);
        break;
      }
    case GDF_SORT:
      {
        // Sort based joins only support single column joins
        if(1 == num_cols)
        {
          gdf_error_code =  sort_join<join_type, output_index_type>(leftcol[0], rightcol[0], left_result, right_result);
        }
        else
        {
          gdf_error_code =  GDF_JOIN_TOO_MANY_COLUMNS;
        }

        break;
      }
    default:
      gdf_error_code =  GDF_UNSUPPORTED_METHOD;
  }

  POP_RANGE();

  return gdf_error_code;
}



template <JoinType join_type, typename index_type>
gdf_error construct_join_output_df(
        std::vector<gdf_column*>& ljoincol,
        std::vector<gdf_column*>& rjoincol,
        gdf_column **left_cols, 
        int num_left_cols,
        int left_join_cols[],
        gdf_column **right_cols,
        int num_right_cols,
        int right_join_cols[],
        int num_cols_to_join,
        int result_num_cols,
        gdf_column ** result_cols,
        gdf_column * left_indices,
        gdf_column * right_indices) {


  PUSH_RANGE("LIBGDF_JOIN_OUTPUT", JOIN_COLOR);
    //create left and right input table with columns not joined on
    std::vector<gdf_column*> lnonjoincol;
    std::vector<gdf_column*> rnonjoincol;
    std::set<int> l_join_indices, r_join_indices;
    for (int i = 0; i < num_cols_to_join; ++i) {
        l_join_indices.insert(left_join_cols[i]);
        r_join_indices.insert(right_join_cols[i]);
    }
    for (int i = 0; i < num_left_cols; ++i) {
        if (l_join_indices.find(i) == l_join_indices.end()) {
            lnonjoincol.push_back(left_cols[i]);
        }
    }
    for (int i = 0; i < num_right_cols; ++i) {
        if (r_join_indices.find(i) == r_join_indices.end()) {
            rnonjoincol.push_back(right_cols[i]);
        }
    }
    //TODO : Invalid api

    gdf_size_type join_size = left_indices->size;
    int left_table_end = num_left_cols - num_cols_to_join;
    int right_table_begin = num_left_cols;

    //create left and right output column data buffers
    for (int i = 0; i < left_table_end; ++i) {
        gdf_column* col = lnonjoincol[i];
        gdf_column_view_augmented(result_cols[i], nullptr, nullptr, join_size,
                                  col->dtype, 0, col->dtype_info, col->col_name);
        int col_width = cudf::byte_width(*(result_cols[i]));
        RMM_TRY( RMM_ALLOC((void**)&(result_cols[i]->data), col_width * join_size, 0) ); // TODO: non-default stream?
        RMM_TRY( RMM_ALLOC((void**)&(result_cols[i]->valid), sizeof(gdf_valid_type)*gdf_valid_allocation_size(join_size), 0) );
        CUDA_TRY( hipMemset(result_cols[i]->valid, 0, sizeof(gdf_valid_type)*gdf_valid_allocation_size(join_size)) );
        CHECK_STREAM(0);
    }
    for (int i = right_table_begin; i < result_num_cols; ++i) {
        gdf_column* col = rnonjoincol[i - right_table_begin];
        gdf_column_view_augmented(result_cols[i], nullptr, nullptr, join_size,
                                  col->dtype, 0, col->dtype_info, col->col_name);
        int col_width = cudf::byte_width(*(result_cols[i]));
        RMM_TRY( RMM_ALLOC((void**)&(result_cols[i]->data), col_width * join_size, 0) ); // TODO: non-default stream?
        RMM_TRY( RMM_ALLOC((void**)&(result_cols[i]->valid), sizeof(gdf_valid_type)*gdf_valid_allocation_size(join_size), 0) );
        CUDA_TRY( hipMemset(result_cols[i]->valid, 0, sizeof(gdf_valid_type)*gdf_valid_allocation_size(join_size)) );
        CHECK_STREAM(0);
    }
    //create joined output column data buffers
    for (int join_index = 0; join_index < num_cols_to_join; ++join_index) {
        int i = left_table_end + join_index;
        gdf_column* col = left_cols[left_join_cols[join_index]];
        gdf_column_view_augmented(result_cols[i], nullptr, nullptr, join_size,
                                  col->dtype, 0, col->dtype_info, col->col_name);
        int col_width = cudf::byte_width(*(result_cols[i]));
        RMM_TRY( RMM_ALLOC((void**)&(result_cols[i]->data), col_width * join_size, 0) ); // TODO: non-default stream?
        RMM_TRY( RMM_ALLOC((void**)&(result_cols[i]->valid), sizeof(gdf_valid_type)*gdf_valid_allocation_size(join_size), 0) );
        CUDA_TRY( hipMemset(result_cols[i]->valid, 0, sizeof(gdf_valid_type)*gdf_valid_allocation_size(join_size)) );
        CHECK_STREAM(0);
    }


    // If the join_type is an outer join, then indices for non-matches will be
    // -1, requiring bounds checking when gathering the result table
    bool const check_bounds{ join_type != JoinType::INNER_JOIN };

    // Construct the left columns
    if (0 != lnonjoincol.size()) {
      cudf::table left_source_table(lnonjoincol.data(), lnonjoincol.size());
      cudf::table left_destination_table(result_cols,
                                         num_left_cols - num_cols_to_join);

      cudf::detail::gather(&left_source_table,
                           static_cast<index_type const *>(left_indices->data),
                           &left_destination_table, check_bounds);
      CHECK_STREAM(0);
      gdf_error update_err = nvcategory_gather_table(left_source_table,left_destination_table);
      CHECK_STREAM(0);
      GDF_REQUIRE(update_err == GDF_SUCCESS,update_err);
    }

    // Construct the right columns
    if (0 != rnonjoincol.size()) {
      cudf::table right_source_table(rnonjoincol.data(), rnonjoincol.size());
      cudf::table right_destination_table(result_cols + right_table_begin,
                                          num_right_cols - num_cols_to_join);

      cudf::detail::gather(&right_source_table,
                           static_cast<index_type const *>(right_indices->data),
                           &right_destination_table, check_bounds);
      CHECK_STREAM(0);
      gdf_error update_err = nvcategory_gather_table(right_source_table,right_destination_table);
      CHECK_STREAM(0);
      GDF_REQUIRE(update_err == GDF_SUCCESS,update_err);
    }

    // Construct the joined columns
    if (0 != ljoincol.size()) {
      cudf::table join_source_table(ljoincol.data(), ljoincol.size());
      cudf::table join_destination_table(result_cols + left_table_end,
                                         num_cols_to_join);

      // Gather valid rows from the right table
      // TODO: Revisit this, because it probably can be done more efficiently
      if (JoinType::FULL_JOIN == join_type) {
        cudf::table right_source_table(rjoincol.data(), rjoincol.size());

        cudf::detail::gather(
            &right_source_table,
            static_cast<index_type const *>(right_indices->data),
            &join_destination_table, check_bounds);
        CHECK_STREAM(0);
      }

      cudf::detail::gather(&join_source_table,
                           static_cast<index_type const *>(left_indices->data),
                           &join_destination_table, check_bounds);
      CHECK_STREAM(0);
      gdf_error update_err = nvcategory_gather_table(join_source_table,join_destination_table);
      CHECK_STREAM(0);
      GDF_REQUIRE(update_err == GDF_SUCCESS,update_err);
    }

    POP_RANGE();
    return GDF_SUCCESS;
}

template <JoinType join_type, typename index_type>
gdf_error join_call_compute_df(
                         gdf_column **left_cols, 
                         int num_left_cols,
                         int left_join_cols[],
                         gdf_column **right_cols,
                         int num_right_cols,
                         int right_join_cols[],
                         int num_cols_to_join,
                         int result_num_cols,
                         gdf_column **result_cols,
                         gdf_column * left_indices,
                         gdf_column * right_indices,
                         gdf_context *join_context) {
  GDF_REQUIRE(nullptr != left_cols, GDF_DATASET_EMPTY);
  GDF_REQUIRE(nullptr != right_cols, GDF_DATASET_EMPTY);
  GDF_REQUIRE(0 != num_cols_to_join, GDF_SUCCESS);
  GDF_REQUIRE(nullptr != left_join_cols, GDF_DATASET_EMPTY);
  GDF_REQUIRE(nullptr != right_join_cols, GDF_DATASET_EMPTY);
  GDF_REQUIRE(nullptr != join_context, GDF_INVALID_API_CALL);

  for(int column_index = 0; column_index  < num_left_cols; column_index++){
    GDF_REQUIRE(left_cols[column_index]->dtype != GDF_invalid,GDF_UNSUPPORTED_DTYPE);
  }
  for(int column_index = 0; column_index  < num_right_cols; column_index++){
    GDF_REQUIRE(right_cols[column_index]->dtype != GDF_invalid,GDF_UNSUPPORTED_DTYPE);
  }

  // Determine if requested output is the indices of matching rows, the fully
  // constructed output dataframe, or both
  bool const construct_output_dataframe{nullptr != result_cols};
  bool const return_output_indices{(nullptr != left_indices) and
                                   (nullptr != right_indices)};

  GDF_REQUIRE(construct_output_dataframe or return_output_indices,
              GDF_INVALID_API_CALL);

  auto const left_col_size = left_cols[0]->size;
  auto const right_col_size = right_cols[0]->size;

  // If the inputs are empty, immediately return
  if ((0 == left_col_size) && (0 == right_col_size)) {
    return GDF_SUCCESS;
  }

  // If left join and the left table is empty, return immediately
  if ((JoinType::LEFT_JOIN == join_type) && (0 == left_col_size)) {
    return GDF_SUCCESS;
  }

  // If Inner Join and either table is empty, return immediately
  if ((JoinType::INNER_JOIN == join_type) &&
      ((0 == left_col_size) || (0 == right_col_size))) {
    return GDF_SUCCESS;
  }


  //if the inputs are nvcategory we need to make the dictionaries comparable
  bool at_least_one_category_column = false;
  for(int join_column_index = 0; join_column_index < num_cols_to_join; join_column_index++){
    at_least_one_category_column |= left_cols[left_join_cols[join_column_index]]->dtype == GDF_STRING_CATEGORY;
  }

  std::vector<gdf_column*> new_left_cols(left_cols, left_cols + num_left_cols);
  std::vector<gdf_column*> new_right_cols(right_cols, right_cols + num_right_cols);
  std::vector<gdf_column *> temp_columns_to_free;
  if(at_least_one_category_column){
    for(int join_column_index = 0; join_column_index < num_cols_to_join; join_column_index++){
      if(left_cols[left_join_cols[join_column_index]]->dtype == GDF_STRING_CATEGORY){
        GDF_REQUIRE(right_cols[right_join_cols[join_column_index]]->dtype == GDF_STRING_CATEGORY, GDF_DTYPE_MISMATCH);

        gdf_column * left_original_column = new_left_cols[left_join_cols[join_column_index]];
        gdf_column * right_original_column = new_right_cols[right_join_cols[join_column_index]];




        gdf_column * new_left_column_ptr = new gdf_column{};
        gdf_column * new_right_column_ptr = new gdf_column{};

        temp_columns_to_free.push_back(new_left_column_ptr);
        temp_columns_to_free.push_back(new_right_column_ptr);


        gdf_column * input_join_columns_merge[2] = {left_original_column, right_original_column};
        gdf_column * new_join_columns[2] = {new_left_column_ptr,
            new_right_column_ptr};
        gdf_column_view(new_left_column_ptr, nullptr, nullptr, left_original_column->size, GDF_STRING_CATEGORY);
        gdf_column_view(new_right_column_ptr, nullptr, nullptr, right_original_column->size, GDF_STRING_CATEGORY);

        int col_width = cudf::byte_width(*new_left_column_ptr);
        RMM_TRY( RMM_ALLOC(&(new_left_column_ptr->data), col_width * left_original_column->size, 0) ); // TODO: non-default stream?
        if(left_original_column->valid != nullptr){
          RMM_TRY( RMM_ALLOC(&(new_left_column_ptr->valid), sizeof(gdf_valid_type)*gdf_valid_allocation_size(left_original_column->size), 0) );
          CUDA_TRY( hipMemcpy(new_left_column_ptr->valid, left_original_column->valid, sizeof(gdf_valid_type)*gdf_num_bitmask_elements(left_original_column->size),hipMemcpyDeviceToDevice) );
        }else{
          new_left_column_ptr->valid = nullptr;
        }
        new_left_column_ptr->null_count = left_original_column->null_count;


        RMM_TRY( RMM_ALLOC(&(new_right_column_ptr->data), col_width * right_original_column->size, 0) ); // TODO: non-default stream?
        if(right_original_column->valid != nullptr){
          RMM_TRY( RMM_ALLOC(&(new_right_column_ptr->valid), sizeof(gdf_valid_type)*gdf_valid_allocation_size(right_original_column->size), 0) );
          CUDA_TRY( hipMemcpy(new_right_column_ptr->valid, right_original_column->valid, sizeof(gdf_valid_type)*gdf_num_bitmask_elements(right_original_column->size),hipMemcpyDeviceToDevice) );
        }else{
          new_right_column_ptr->valid = nullptr;
        }
        new_right_column_ptr->null_count = right_original_column->null_count;
        gdf_error err = sync_column_categories(input_join_columns_merge,
            new_join_columns,
            2);

        GDF_REQUIRE(GDF_SUCCESS == err, err);

        new_left_cols[left_join_cols[join_column_index]] = new_join_columns[0];
        new_right_cols[right_join_cols[join_column_index]] = new_join_columns[1];
        CHECK_STREAM(0);
      }
    }


    left_cols = new_left_cols.data();
    right_cols = new_right_cols.data();
  }

  // If index outputs are not requested, create columns to store them
  // for computing combined join output
  gdf_column *left_index_out = left_indices;
  gdf_column *right_index_out = right_indices;

  using gdf_col_pointer =
      typename std::unique_ptr<gdf_column, std::function<void(gdf_column *)>>;
  auto gdf_col_deleter = [](gdf_column *col) {
    col->size = 0;
    if (col->data) {
      RMM_FREE(col->data, 0);
    }
    if (col->valid) {
      RMM_FREE(col->valid, 0);
    }
  };
  gdf_col_pointer l_index_temp, r_index_temp;

  if (nullptr == left_indices) {
    l_index_temp = {new gdf_column{}, gdf_col_deleter};
    left_index_out = l_index_temp.get();
    }

    if (nullptr == right_indices) {
        r_index_temp = {new gdf_column{}, gdf_col_deleter};
        right_index_out = r_index_temp.get();
    }

    //get column pointers to join on
    std::vector<gdf_column*> ljoincol;
    std::vector<gdf_column*> rjoincol;
    for (int i = 0; i < num_cols_to_join; ++i) {
        ljoincol.push_back(left_cols[ left_join_cols[i] ]);
        rjoincol.push_back(right_cols[ right_join_cols[i] ]);
    }


    gdf_error join_err = join_call<join_type>(num_cols_to_join,
            ljoincol.data(), rjoincol.data(),
            left_index_out, right_index_out,
            join_context);
    CHECK_STREAM(0);
    GDF_REQUIRE(GDF_SUCCESS == join_err, join_err);

    //If construct_output_dataframe is false then left_index_out or right_index_out
    //was not dynamically allocated.
    if (not construct_output_dataframe) {
        return join_err;
    }

    gdf_error df_err =
        construct_join_output_df<join_type, index_type>(
            ljoincol, rjoincol,
            left_cols, num_left_cols, left_join_cols,
            right_cols, num_right_cols, right_join_cols,
            num_cols_to_join, result_num_cols, result_cols,
            left_index_out, right_index_out);
    CHECK_STREAM(0);
    l_index_temp.reset(nullptr);
    r_index_temp.reset(nullptr);





    //freeing up the temp column used to synch categories between columns
    for(unsigned int column_to_free = 0; column_to_free < temp_columns_to_free.size(); column_to_free++){
      gdf_column_free(temp_columns_to_free[column_to_free]);
      delete temp_columns_to_free[column_to_free];
    }

    CHECK_STREAM(0);


    return df_err;
}

gdf_error gdf_left_join(
                         gdf_column **left_cols, 
                         int num_left_cols,
                         int left_join_cols[],
                         gdf_column **right_cols,
                         int num_right_cols,
                         int right_join_cols[],
                         int num_cols_to_join,
                         int result_num_cols,
                         gdf_column **result_cols,
                         gdf_column * left_indices,
                         gdf_column * right_indices,
                         gdf_context *join_context) {
    return join_call_compute_df<JoinType::LEFT_JOIN, output_index_type>(
                     left_cols, 
                     num_left_cols,
                     left_join_cols,
                     right_cols,
                     num_right_cols,
                     right_join_cols,
                     num_cols_to_join,
                     result_num_cols,
                     result_cols,
                     left_indices,
                     right_indices,
                     join_context);
}

gdf_error gdf_inner_join(
                         gdf_column **left_cols, 
                         int num_left_cols,
                         int left_join_cols[],
                         gdf_column **right_cols,
                         int num_right_cols,
                         int right_join_cols[],
                         int num_cols_to_join,
                         int result_num_cols,
                         gdf_column **result_cols,
                         gdf_column * left_indices,
                         gdf_column * right_indices,
                         gdf_context *join_context) {
    return join_call_compute_df<JoinType::INNER_JOIN, output_index_type>(
                     left_cols, 
                     num_left_cols,
                     left_join_cols,
                     right_cols,
                     num_right_cols,
                     right_join_cols,
                     num_cols_to_join,
                     result_num_cols,
                     result_cols,
                     left_indices,
                     right_indices,
                     join_context);
}

gdf_error gdf_full_join(
                         gdf_column **left_cols, 
                         int num_left_cols,
                         int left_join_cols[],
                         gdf_column **right_cols,
                         int num_right_cols,
                         int right_join_cols[],
                         int num_cols_to_join,
                         int result_num_cols,
                         gdf_column **result_cols,
                         gdf_column * left_indices,
                         gdf_column * right_indices,
                         gdf_context *join_context) {
    return join_call_compute_df<JoinType::FULL_JOIN, output_index_type>(
                     left_cols, 
                     num_left_cols,
                     left_join_cols,
                     right_cols,
                     num_right_cols,
                     right_join_cols,
                     num_cols_to_join,
                     result_num_cols,
                     result_cols,
                     left_indices,
                     right_indices,
                     join_context);
}
