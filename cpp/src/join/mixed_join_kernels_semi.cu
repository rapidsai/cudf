#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2022-2024, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "join/mixed_join_kernels_semi.cuh"

#include <cudf/ast/detail/expression_evaluator.cuh>
#include <cudf/ast/detail/expression_parser.hpp>
#include <cudf/detail/utilities/cuda.cuh>
#include <cudf/table/table_device_view.cuh>
#include <cudf/utilities/export.hpp>
#include <cudf/utilities/span.hpp>

#include <hipcub/hipcub.hpp>

namespace cudf {
namespace detail {

namespace cg = cooperative_groups;

#pragma GCC diagnostic ignored "-Wattributes"

template <cudf::size_type block_size, bool has_nulls>
CUDF_KERNEL void __launch_bounds__(block_size)
  mixed_join_semi(table_device_view left_table,
                  table_device_view right_table,
                  table_device_view probe,
                  table_device_view build,
                  row_equality const equality_probe,
                  hash_set_ref_type set_ref,
                  cudf::device_span<bool> left_table_keep_mask,
                  cudf::ast::detail::expression_device_view device_expression_data)
{
  auto constexpr cg_size = hash_set_ref_type::cg_size;

  auto const tile = cg::tiled_partition<cg_size>(cg::this_thread_block());

  // Normally the casting of a shared memory array is used to create multiple
  // arrays of different types from the shared memory buffer, but here it is
  // used to circumvent conflicts between arrays of different types between
  // different template instantiations due to the extern specifier.
  extern __shared__ char raw_intermediate_storage[];
  auto intermediate_storage =
    reinterpret_cast<cudf::ast::detail::IntermediateDataType<has_nulls>*>(raw_intermediate_storage);
  auto thread_intermediate_storage =
    intermediate_storage + (tile.meta_group_rank() * device_expression_data.num_intermediates);

  // Equality evaluator to use
  auto const evaluator = cudf::ast::detail::expression_evaluator<has_nulls>(
    left_table, right_table, device_expression_data);

  // Make sure to swap_tables here as hash_set will use probe table as the left one
  auto constexpr swap_tables = true;
  auto const equality        = single_expression_equality<has_nulls>{
    evaluator, thread_intermediate_storage, swap_tables, equality_probe};

  // Create set ref with the new equality comparator
  auto const set_ref_equality = set_ref.with_key_eq(equality);

  // Total number of rows to query the set
  auto const outer_num_rows = left_table.num_rows();
  // Grid stride for the tile
  auto const cg_grid_stride = cudf::detail::grid_1d::grid_stride<block_size>() / cg_size;

  // Find all the rows in the left table that are in the hash table
  for (auto outer_row_index = cudf::detail::grid_1d::global_thread_id<block_size>() / cg_size;
       outer_row_index < outer_num_rows;
       outer_row_index += cg_grid_stride) {
    auto const result = set_ref_equality.contains(tile, outer_row_index);
    if (tile.thread_rank() == 0) { left_table_keep_mask[outer_row_index] = result; }
  }
}

void launch_mixed_join_semi(bool has_nulls,
                            table_device_view left_table,
                            table_device_view right_table,
                            table_device_view probe,
                            table_device_view build,
                            row_equality const equality_probe,
                            hash_set_ref_type set_ref,
                            cudf::device_span<bool> left_table_keep_mask,
                            cudf::ast::detail::expression_device_view device_expression_data,
                            detail::grid_1d const config,
                            int64_t shmem_size_per_block,
                            rmm::cuda_stream_view stream)
{
  if (has_nulls) {
    mixed_join_semi<DEFAULT_JOIN_BLOCK_SIZE, true>
      <<<config.num_blocks, config.num_threads_per_block, shmem_size_per_block, stream.value()>>>(
        left_table,
        right_table,
        probe,
        build,
        equality_probe,
        set_ref,
        left_table_keep_mask,
        device_expression_data);
  } else {
    mixed_join_semi<DEFAULT_JOIN_BLOCK_SIZE, false>
      <<<config.num_blocks, config.num_threads_per_block, shmem_size_per_block, stream.value()>>>(
        left_table,
        right_table,
        probe,
        build,
        equality_probe,
        set_ref,
        left_table_keep_mask,
        device_expression_data);
  }
}

}  // namespace detail
}  // namespace cudf
