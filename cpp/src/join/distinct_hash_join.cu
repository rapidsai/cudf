#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2024, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */
#include "join_common_utils.cuh"
#include "join_common_utils.hpp"

#include <cudf/detail/cuco_helpers.hpp>
#include <cudf/detail/distinct_hash_join.cuh>
#include <cudf/detail/nvtx/ranges.hpp>
#include <cudf/detail/utilities/cuda.cuh>
#include <cudf/join.hpp>
#include <cudf/table/experimental/row_operators.cuh>
#include <cudf/table/table_view.hpp>
#include <cudf/types.hpp>
#include <cudf/utilities/memory_resource.hpp>

#include <rmm/cuda_stream_view.hpp>
#include <rmm/device_scalar.hpp>
#include <rmm/device_uvector.hpp>

#include <hip/hip_cooperative_groups.h>
#include <cub/block/block_scan.cuh>
#include <cuco/static_set.cuh>
#include <thrust/fill.h>
#include <thrust/iterator/transform_output_iterator.h>
#include <thrust/sequence.h>

#include <cstddef>
#include <limits>
#include <memory>
#include <utility>
#include <variant>

namespace cudf {
namespace detail {
namespace {

auto prepare_device_equal(
  std::shared_ptr<cudf::experimental::row::equality::preprocessed_table> build,
  std::shared_ptr<cudf::experimental::row::equality::preprocessed_table> probe,
  bool has_nulls,
  cudf::null_equality compare_nulls,
  std::unordered_set<cudf::type_id> const& column_types)
{
  auto const two_table_equal =
    cudf::experimental::row::equality::two_table_comparator(probe, build);
  auto d_comparator =
    two_table_equal.equal_to(column_types, nullate::DYNAMIC{has_nulls}, compare_nulls);

  // need to wrap types in d_comparator with comparator_adaptor. We can do this with some
  // metaprogramming but this seems fine for now
  using ret_type = std::variant<
    comparator_adapter<cudf::experimental::row::equality::strong_index_comparator_adapter<
      cudf::experimental::row::equality::device_row_comparator<
        true,
        nullate::DYNAMIC,
        cudf::experimental::row::equality::nan_equal_physical_equality_comparator,
        cudf::experimental::type_identity_t>>>,
    comparator_adapter<cudf::experimental::row::equality::strong_index_comparator_adapter<
      cudf::experimental::row::equality::device_row_comparator<
        false,
        nullate::DYNAMIC,
        cudf::experimental::row::equality::nan_equal_physical_equality_comparator,
        cudf::experimental::dispatch_void_if_nested_t>>>,
    comparator_adapter<cudf::experimental::row::equality::strong_index_comparator_adapter<
      cudf::experimental::row::equality::device_row_comparator<
        false,
        nullate::DYNAMIC,
        cudf::experimental::row::equality::nan_equal_physical_equality_comparator,
        cudf::experimental::dispatch_void_if_compound_t>>>>;

  return std::visit(
    [&](auto& comparator) {
      return ret_type{std::in_place_type<
                        comparator_adapter<typename std::remove_reference_t<decltype(comparator)>>>,
                      comparator};
    },
    d_comparator);
}

/**
 * @brief Device functor to create a pair of {hash_value, row_index} for a given row.
 *
 * @tparam Hasher The type of internal hasher to compute row hash.
 */
template <typename Hasher, typename T>
class build_keys_fn {
 public:
  CUDF_HOST_DEVICE build_keys_fn(Hasher const& hash) : _hash{hash} {}

  __device__ __forceinline__ auto operator()(size_type i) const noexcept
  {
    return cuco::pair{_hash(i), T{i}};
  }

 private:
  Hasher _hash;
};

/**
 * @brief Device output transform functor to construct `size_type` with
 * `cuco::pair<hash_value_type, lhs_index_type>` or `cuco::pair<hash_value_type, rhs_index_type>`
 */
struct output_fn {
  __device__ constexpr cudf::size_type operator()(
    cuco::pair<hash_value_type, lhs_index_type> const& x) const
  {
    return static_cast<cudf::size_type>(x.second);
  }
  __device__ constexpr cudf::size_type operator()(
    cuco::pair<hash_value_type, rhs_index_type> const& x) const
  {
    return static_cast<cudf::size_type>(x.second);
  }
};
}  // namespace

template <cudf::has_nested HasNested>
distinct_hash_join<HasNested>::distinct_hash_join(cudf::table_view const& build,
                                                  cudf::table_view const& probe,
                                                  bool has_nulls,
                                                  cudf::null_equality compare_nulls,
                                                  rmm::cuda_stream_view stream)
  : _has_nulls{has_nulls},
    _nulls_equal{compare_nulls},
    _build{build},
    _probe{probe},
    _preprocessed_build{
      cudf::experimental::row::equality::preprocessed_table::create(_build, stream)},
    _preprocessed_probe{
      cudf::experimental::row::equality::preprocessed_table::create(_probe, stream)}
{
  std::unordered_set<cudf::type_id> build_column_types;
  for (auto col : this->_build) {
    build_column_types.insert(col.type().id());
  }

  auto var_comparator_adapter = prepare_device_equal(
    _preprocessed_build, _preprocessed_probe, has_nulls, compare_nulls, build_column_types);

  std::visit(
    [&](auto&& comparator_adapter) {
      using static_set_type =
        cuco::static_set<cuco::pair<hash_value_type, rhs_index_type>,
                         cuco::extent<size_type>,
                         cuda::thread_scope_device,
                         typename std::remove_reference_t<decltype(comparator_adapter)>,
                         distinct_hash_join::probing_scheme_type,
                         cudf::detail::cuco_allocator<char>,
                         distinct_hash_join::cuco_storage_type>;

      this->_hash_table = std::make_unique<hash_table_type>(
        std::in_place_type<static_set_type>,
        build.num_rows(),
        CUCO_DESIRED_LOAD_FACTOR,
        cuco::empty_key{
          cuco::pair{std::numeric_limits<hash_value_type>::max(), rhs_index_type{JoinNoneValue}}},
        comparator_adapter,
        distinct_hash_join::probing_scheme_type{},
        cuco::thread_scope_device,
        distinct_hash_join::cuco_storage_type{},
        cudf::detail::cuco_allocator<char>{rmm::mr::polymorphic_allocator<char>{}, stream},
        stream.value());
    },
    var_comparator_adapter);

  CUDF_FUNC_RANGE();
  CUDF_EXPECTS(0 != this->_build.num_columns(), "Hash join build table is empty");

  if (this->_build.num_rows() == 0) { return; }

  auto const row_hasher = experimental::row::hash::row_hasher{this->_preprocessed_build};
  auto const d_hasher =
    row_hasher.device_hasher(build_column_types, nullate::DYNAMIC{this->_has_nulls});

  std::visit(
    [&](auto&& hasher, auto&& hash_table) {
      auto const iter = cudf::detail::make_counting_transform_iterator(
        0,
        build_keys_fn<typename std::remove_reference_t<decltype(hasher)>, rhs_index_type>{hasher});

      size_type const build_table_num_rows{build.num_rows()};
      if (this->_nulls_equal == cudf::null_equality::EQUAL or (not cudf::nullable(this->_build))) {
        hash_table.insert_async(iter, iter + build_table_num_rows, stream.value());
      } else {
        auto stencil = thrust::counting_iterator<size_type>{0};
        auto const row_bitmask =
          cudf::detail::bitmask_and(this->_build, stream, cudf::get_current_device_resource_ref())
            .first;
        auto const pred =
          cudf::detail::row_is_valid{reinterpret_cast<bitmask_type const*>(row_bitmask.data())};

        // insert valid rows
        hash_table.insert_if_async(
          iter, iter + build_table_num_rows, stencil, pred, stream.value());
      }
    },
    d_hasher,
    *(this->_hash_table));
}

template <cudf::has_nested HasNested>
std::pair<std::unique_ptr<rmm::device_uvector<size_type>>,
          std::unique_ptr<rmm::device_uvector<size_type>>>
distinct_hash_join<HasNested>::inner_join(rmm::cuda_stream_view stream,
                                          rmm::device_async_resource_ref mr) const
{
  cudf::scoped_range range{"distinct_hash_join::inner_join"};

  size_type const probe_table_num_rows{this->_probe.num_rows()};

  // If output size is zero, return immediately
  if (probe_table_num_rows == 0) {
    return std::pair(std::make_unique<rmm::device_uvector<size_type>>(0, stream, mr),
                     std::make_unique<rmm::device_uvector<size_type>>(0, stream, mr));
  }

  auto build_indices =
    std::make_unique<rmm::device_uvector<size_type>>(probe_table_num_rows, stream, mr);
  auto probe_indices =
    std::make_unique<rmm::device_uvector<size_type>>(probe_table_num_rows, stream, mr);

  std::unordered_set<cudf::type_id> probe_column_types;
  for (auto col : this->_probe) {
    probe_column_types.insert(col.type().id());
  }

  auto const probe_row_hasher =
    cudf::experimental::row::hash::row_hasher{this->_preprocessed_probe};
  auto const d_probe_hasher =
    probe_row_hasher.device_hasher(probe_column_types, nullate::DYNAMIC{this->_has_nulls});

  auto const build_indices_begin =
    thrust::make_transform_output_iterator(build_indices->begin(), output_fn{});
  auto const probe_indices_begin =
    thrust::make_transform_output_iterator(probe_indices->begin(), output_fn{});

  std::visit(
    [&](auto&& hasher, auto&& hash_table) {
      auto const iter = cudf::detail::make_counting_transform_iterator(
        0,
        build_keys_fn<typename std::remove_reference_t<decltype(hasher)>, lhs_index_type>{hasher});

      auto const [probe_indices_end, _] = hash_table.retrieve(iter,
                                                              iter + probe_table_num_rows,
                                                              probe_indices_begin,
                                                              build_indices_begin,
                                                              {stream.value()});

      auto const actual_size = std::distance(probe_indices_begin, probe_indices_end);
      build_indices->resize(actual_size, stream);
      probe_indices->resize(actual_size, stream);
    },
    d_probe_hasher,
    *(this->_hash_table));

  return {std::move(build_indices), std::move(probe_indices)};
}

template <cudf::has_nested HasNested>
std::unique_ptr<rmm::device_uvector<size_type>> distinct_hash_join<HasNested>::left_join(
  rmm::cuda_stream_view stream, rmm::device_async_resource_ref mr) const
{
  cudf::scoped_range range{"distinct_hash_join::left_join"};

  size_type const probe_table_num_rows{this->_probe.num_rows()};

  // If output size is zero, return empty
  if (probe_table_num_rows == 0) {
    return std::make_unique<rmm::device_uvector<size_type>>(0, stream, mr);
  }

  std::unordered_set<cudf::type_id> probe_column_types;
  for (auto col : _probe) {
    probe_column_types.insert(col.type().id());
  }

  auto build_indices =
    std::make_unique<rmm::device_uvector<size_type>>(probe_table_num_rows, stream, mr);

  // If build table is empty, return probe table
  if (this->_build.num_rows() == 0) {
    thrust::fill(
      rmm::exec_policy_nosync(stream), build_indices->begin(), build_indices->end(), JoinNoneValue);
  } else {
    auto const probe_row_hasher =
      cudf::experimental::row::hash::row_hasher{this->_preprocessed_probe};
    auto const d_probe_hasher =
      probe_row_hasher.device_hasher(probe_column_types, nullate::DYNAMIC{this->_has_nulls});

    std::visit(
      [&](auto&& hasher, auto&& hash_table) {
        auto const iter = cudf::detail::make_counting_transform_iterator(
          0,
          build_keys_fn<typename std::remove_reference_t<decltype(hasher)>, lhs_index_type>{
            hasher});

        auto const output_begin =
          thrust::make_transform_output_iterator(build_indices->begin(), output_fn{});
        // TODO conditional find for nulls once `cuco::static_set::find_if` is added
        hash_table.find_async(iter, iter + probe_table_num_rows, output_begin, stream.value());
      },
      d_probe_hasher,
      *(this->_hash_table));
  }

  return build_indices;
}
}  // namespace detail

template <>
distinct_hash_join<cudf::has_nested::YES>::~distinct_hash_join() = default;

template <>
distinct_hash_join<cudf::has_nested::NO>::~distinct_hash_join() = default;

template <>
distinct_hash_join<cudf::has_nested::YES>::distinct_hash_join(cudf::table_view const& build,
                                                              cudf::table_view const& probe,
                                                              nullable_join has_nulls,
                                                              null_equality compare_nulls,
                                                              rmm::cuda_stream_view stream)
  : _impl{std::make_unique<impl_type>(
      build, probe, has_nulls == nullable_join::YES, compare_nulls, stream)}
{
}

template <>
distinct_hash_join<cudf::has_nested::NO>::distinct_hash_join(cudf::table_view const& build,
                                                             cudf::table_view const& probe,
                                                             nullable_join has_nulls,
                                                             null_equality compare_nulls,
                                                             rmm::cuda_stream_view stream)
  : _impl{std::make_unique<impl_type>(
      build, probe, has_nulls == nullable_join::YES, compare_nulls, stream)}
{
}

template <>
std::pair<std::unique_ptr<rmm::device_uvector<size_type>>,
          std::unique_ptr<rmm::device_uvector<size_type>>>
distinct_hash_join<cudf::has_nested::YES>::inner_join(rmm::cuda_stream_view stream,
                                                      rmm::device_async_resource_ref mr) const
{
  return _impl->inner_join(stream, mr);
}

template <>
std::pair<std::unique_ptr<rmm::device_uvector<size_type>>,
          std::unique_ptr<rmm::device_uvector<size_type>>>
distinct_hash_join<cudf::has_nested::NO>::inner_join(rmm::cuda_stream_view stream,
                                                     rmm::device_async_resource_ref mr) const
{
  return _impl->inner_join(stream, mr);
}

template <>
std::unique_ptr<rmm::device_uvector<size_type>>
distinct_hash_join<cudf::has_nested::YES>::left_join(rmm::cuda_stream_view stream,
                                                     rmm::device_async_resource_ref mr) const
{
  return _impl->left_join(stream, mr);
}

template <>
std::unique_ptr<rmm::device_uvector<size_type>> distinct_hash_join<cudf::has_nested::NO>::left_join(
  rmm::cuda_stream_view stream, rmm::device_async_resource_ref mr) const
{
  return _impl->left_join(stream, mr);
}
}  // namespace cudf
