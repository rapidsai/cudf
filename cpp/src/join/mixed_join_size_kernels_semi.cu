#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2022, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <join/join_common_utils.cuh>
#include <join/join_common_utils.hpp>
#include <join/mixed_join_common_utils.cuh>

#include <cudf/ast/detail/expression_evaluator.cuh>
#include <cudf/ast/detail/expression_parser.hpp>
#include <cudf/detail/utilities/cuda.cuh>
#include <cudf/table/table_device_view.cuh>
#include <cudf/utilities/span.hpp>

#include <hipcub/hipcub.hpp>

namespace cudf {
namespace detail {

namespace cg = cooperative_groups;

template <int block_size, bool has_nulls>
__launch_bounds__(block_size) __global__ void compute_mixed_join_output_size_semi(
  table_device_view left_table,
  table_device_view right_table,
  table_device_view probe,
  table_device_view build,
  row_equality const equality_probe,
  join_kind const join_type,
  cudf::detail::semi_map_type::device_view hash_table_view,
  ast::detail::expression_device_view device_expression_data,
  bool const swap_tables,
  std::size_t* output_size,
  cudf::device_span<cudf::size_type> matches_per_row)
{
  // The (required) extern storage of the shared memory array leads to
  // conflicting declarations between different templates. The easiest
  // workaround is to declare an arbitrary (here char) array type then cast it
  // after the fact to the appropriate type.
  extern __shared__ char raw_intermediate_storage[];
  cudf::ast::detail::IntermediateDataType<has_nulls>* intermediate_storage =
    reinterpret_cast<cudf::ast::detail::IntermediateDataType<has_nulls>*>(raw_intermediate_storage);
  auto thread_intermediate_storage =
    intermediate_storage + (threadIdx.x * device_expression_data.num_intermediates);

  std::size_t thread_counter{0};
  cudf::size_type const start_idx      = threadIdx.x + blockIdx.x * block_size;
  cudf::size_type const stride         = block_size * gridDim.x;
  cudf::size_type const left_num_rows  = left_table.num_rows();
  cudf::size_type const right_num_rows = right_table.num_rows();
  auto const outer_num_rows            = (swap_tables ? right_num_rows : left_num_rows);

  auto evaluator = cudf::ast::detail::expression_evaluator<has_nulls>(
    left_table, right_table, device_expression_data);
  row_hash hash_probe{nullate::DYNAMIC{has_nulls}, probe};
  // TODO: Address asymmetry in operator.
  auto equality = single_expression_equality<has_nulls>{
    evaluator, thread_intermediate_storage, swap_tables, equality_probe};

  for (cudf::size_type outer_row_index = start_idx; outer_row_index < outer_num_rows;
       outer_row_index += stride) {
    matches_per_row[outer_row_index] =
      ((join_type == join_kind::LEFT_ANTI_JOIN) !=
       (hash_table_view.contains(outer_row_index, hash_probe, equality)));
    thread_counter += matches_per_row[outer_row_index];
  }

  using BlockReduce = hipcub::BlockReduce<cudf::size_type, block_size>;
  __shared__ typename BlockReduce::TempStorage temp_storage;
  std::size_t block_counter = BlockReduce(temp_storage).Sum(thread_counter);

  // Add block counter to global counter
  if (threadIdx.x == 0) atomicAdd(output_size, block_counter);
}

template __global__ void compute_mixed_join_output_size_semi<DEFAULT_JOIN_BLOCK_SIZE, true>(
  table_device_view left_table,
  table_device_view right_table,
  table_device_view probe,
  table_device_view build,
  row_equality const equality_probe,
  join_kind const join_type,
  cudf::detail::semi_map_type::device_view hash_table_view,
  ast::detail::expression_device_view device_expression_data,
  bool const swap_tables,
  std::size_t* output_size,
  cudf::device_span<cudf::size_type> matches_per_row);

template __global__ void compute_mixed_join_output_size_semi<DEFAULT_JOIN_BLOCK_SIZE, false>(
  table_device_view left_table,
  table_device_view right_table,
  table_device_view probe,
  table_device_view build,
  row_equality const equality_probe,
  join_kind const join_type,
  cudf::detail::semi_map_type::device_view hash_table_view,
  ast::detail::expression_device_view device_expression_data,
  bool const swap_tables,
  std::size_t* output_size,
  cudf::device_span<cudf::size_type> matches_per_row);

}  // namespace detail

}  // namespace cudf
