/*
 * Copyright (c) 2019-2022, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */
#include "join_common_utils.hpp"

#include <cudf/detail/gather.cuh>
#include <cudf/dictionary/detail/update_keys.hpp>
#include <cudf/join.hpp>
#include <cudf/table/table.hpp>
#include <cudf/table/table_view.hpp>
#include <cudf/utilities/default_stream.hpp>

#include <rmm/cuda_stream_view.hpp>

namespace cudf {
namespace detail {

std::pair<std::unique_ptr<rmm::device_uvector<size_type>>,
          std::unique_ptr<rmm::device_uvector<size_type>>>
inner_join(table_view const& left_input,
           table_view const& right_input,
           null_equality compare_nulls,
           rmm::cuda_stream_view stream,
           rmm::mr::device_memory_resource* mr)
{
  // Make sure any dictionary columns have matched key sets.
  // This will return any new dictionary columns created as well as updated table_views.
  auto matched = cudf::dictionary::detail::match_dictionaries(
    {left_input, right_input},
    stream,
    rmm::mr::get_current_device_resource());  // temporary objects returned

  // now rebuild the table views with the updated ones
  auto const left  = matched.second.front();
  auto const right = matched.second.back();

  // For `inner_join`, we can freely choose either the `left` or `right` table to use for
  // building/probing the hash map. Because building is typically more expensive than probing, we
  // build the hash map from the smaller table.
  if (right.num_rows() > left.num_rows()) {
    cudf::hash_join hj_obj(left, compare_nulls, stream);
    auto [right_result, left_result] = hj_obj.inner_join(right, std::nullopt, stream, mr);
    return std::pair(std::move(left_result), std::move(right_result));
  } else {
    cudf::hash_join hj_obj(right, compare_nulls, stream);
    return hj_obj.inner_join(left, std::nullopt, stream, mr);
  }
}

std::pair<std::unique_ptr<rmm::device_uvector<size_type>>,
          std::unique_ptr<rmm::device_uvector<size_type>>>
left_join(table_view const& left_input,
          table_view const& right_input,
          null_equality compare_nulls,
          rmm::cuda_stream_view stream,
          rmm::mr::device_memory_resource* mr)
{
  // Make sure any dictionary columns have matched key sets.
  // This will return any new dictionary columns created as well as updated table_views.
  auto matched = cudf::dictionary::detail::match_dictionaries(
    {left_input, right_input},  // these should match
    stream,
    rmm::mr::get_current_device_resource());  // temporary objects returned
  // now rebuild the table views with the updated ones
  table_view const left  = matched.second.front();
  table_view const right = matched.second.back();

  cudf::hash_join hj_obj(right, compare_nulls, stream);
  return hj_obj.left_join(left, std::nullopt, stream, mr);
}

std::pair<std::unique_ptr<rmm::device_uvector<size_type>>,
          std::unique_ptr<rmm::device_uvector<size_type>>>
full_join(table_view const& left_input,
          table_view const& right_input,
          null_equality compare_nulls,
          rmm::cuda_stream_view stream,
          rmm::mr::device_memory_resource* mr)
{
  // Make sure any dictionary columns have matched key sets.
  // This will return any new dictionary columns created as well as updated table_views.
  auto matched = cudf::dictionary::detail::match_dictionaries(
    {left_input, right_input},  // these should match
    stream,
    rmm::mr::get_current_device_resource());  // temporary objects returned
  // now rebuild the table views with the updated ones
  table_view const left  = matched.second.front();
  table_view const right = matched.second.back();

  cudf::hash_join hj_obj(right, compare_nulls, stream);
  return hj_obj.full_join(left, std::nullopt, stream, mr);
}

}  // namespace detail

std::pair<std::unique_ptr<rmm::device_uvector<size_type>>,
          std::unique_ptr<rmm::device_uvector<size_type>>>
inner_join(table_view const& left,
           table_view const& right,
           null_equality compare_nulls,
           rmm::mr::device_memory_resource* mr)
{
  CUDF_FUNC_RANGE();
  return detail::inner_join(left, right, compare_nulls, cudf::get_default_stream(), mr);
}

std::pair<std::unique_ptr<rmm::device_uvector<size_type>>,
          std::unique_ptr<rmm::device_uvector<size_type>>>
left_join(table_view const& left,
          table_view const& right,
          null_equality compare_nulls,
          rmm::mr::device_memory_resource* mr)
{
  CUDF_FUNC_RANGE();
  return detail::left_join(left, right, compare_nulls, cudf::get_default_stream(), mr);
}

std::pair<std::unique_ptr<rmm::device_uvector<size_type>>,
          std::unique_ptr<rmm::device_uvector<size_type>>>
full_join(table_view const& left,
          table_view const& right,
          null_equality compare_nulls,
          rmm::mr::device_memory_resource* mr)
{
  CUDF_FUNC_RANGE();
  return detail::full_join(left, right, compare_nulls, cudf::get_default_stream(), mr);
}

}  // namespace cudf
