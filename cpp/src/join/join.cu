/*
 * Copyright (c) 2019-2020, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */
#include <join/hash_join.cuh>
#include <join/join_common_utils.hpp>

#include <cudf/detail/nvtx/ranges.hpp>
#include <cudf/dictionary/detail/update_keys.hpp>
#include <cudf/join.hpp>
#include <cudf/table/table.hpp>

#include <rmm/cuda_stream_view.hpp>

namespace cudf {
namespace detail {

std::pair<std::unique_ptr<rmm::device_uvector<size_type>>,
          std::unique_ptr<rmm::device_uvector<size_type>>>
inner_join(table_view const& left_input,
           table_view const& right_input,
           null_equality compare_nulls,
           rmm::cuda_stream_view stream,
           rmm::mr::device_memory_resource* mr)
{
  // Make sure any dictionary columns have matched key sets.
  // This will return any new dictionary columns created as well as updated table_views.
  auto matched = cudf::dictionary::detail::match_dictionaries(
    {left_input, right_input},
    stream,
    rmm::mr::get_current_device_resource());  // temporary objects returned

  // now rebuild the table views with the updated ones
  auto const left  = matched.second.front();
  auto const right = matched.second.back();

  // For `inner_join`, we can freely choose either the `left` or `right` table to use for
  // building/probing the hash map. Because building is typically more expensive than probing, we
  // build the hash map from the smaller table.
  if (right.num_rows() > left.num_rows()) {
    cudf::hash_join hj_obj(left, compare_nulls, stream);
    auto result = hj_obj.inner_join(right, compare_nulls, std::nullopt, stream, mr);
    return std::make_pair(std::move(result.second), std::move(result.first));
  } else {
    cudf::hash_join hj_obj(right, compare_nulls, stream);
    return hj_obj.inner_join(left, compare_nulls, std::nullopt, stream, mr);
  }
}

std::unique_ptr<table> inner_join(table_view const& left_input,
                                  table_view const& right_input,
                                  std::vector<size_type> const& left_on,
                                  std::vector<size_type> const& right_on,
                                  null_equality compare_nulls,
                                  rmm::cuda_stream_view stream,
                                  rmm::mr::device_memory_resource* mr)
{
  // Make sure any dictionary columns have matched key sets.
  // This will return any new dictionary columns created as well as updated table_views.
  auto matched = cudf::dictionary::detail::match_dictionaries(
    {left_input.select(left_on), right_input.select(right_on)},
    stream,
    rmm::mr::get_current_device_resource());  // temporary objects returned

  // now rebuild the table views with the updated ones
  auto const left  = scatter_columns(matched.second.front(), left_on, left_input);
  auto const right = scatter_columns(matched.second.back(), right_on, right_input);

  auto join_indices = inner_join(left.select(left_on), right.select(right_on), compare_nulls, mr);
  std::unique_ptr<table> left_result  = detail::gather(left,
                                                      join_indices.first->begin(),
                                                      join_indices.first->end(),
                                                      out_of_bounds_policy::DONT_CHECK,
                                                      stream,
                                                      mr);
  std::unique_ptr<table> right_result = detail::gather(right,
                                                       join_indices.second->begin(),
                                                       join_indices.second->end(),
                                                       out_of_bounds_policy::DONT_CHECK,
                                                       stream,
                                                       mr);
  return combine_table_pair(std::move(left_result), std::move(right_result));
}

std::pair<std::unique_ptr<rmm::device_uvector<size_type>>,
          std::unique_ptr<rmm::device_uvector<size_type>>>
left_join(table_view const& left_input,
          table_view const& right_input,
          null_equality compare_nulls,
          rmm::cuda_stream_view stream,
          rmm::mr::device_memory_resource* mr)
{
  // Make sure any dictionary columns have matched key sets.
  // This will return any new dictionary columns created as well as updated table_views.
  auto matched = cudf::dictionary::detail::match_dictionaries(
    {left_input, right_input},  // these should match
    stream,
    rmm::mr::get_current_device_resource());  // temporary objects returned
  // now rebuild the table views with the updated ones
  table_view const left  = matched.second.front();
  table_view const right = matched.second.back();

  cudf::hash_join hj_obj(right, compare_nulls, stream);
  return hj_obj.left_join(left, compare_nulls, std::nullopt, stream, mr);
}

std::unique_ptr<table> left_join(table_view const& left_input,
                                 table_view const& right_input,
                                 std::vector<size_type> const& left_on,
                                 std::vector<size_type> const& right_on,
                                 null_equality compare_nulls,
                                 rmm::cuda_stream_view stream,
                                 rmm::mr::device_memory_resource* mr)
{
  // Make sure any dictionary columns have matched key sets.
  // This will return any new dictionary columns created as well as updated table_views.
  auto matched = cudf::dictionary::detail::match_dictionaries(
    {left_input.select(left_on), right_input.select(right_on)},  // these should match
    stream,
    rmm::mr::get_current_device_resource());  // temporary objects returned
  // now rebuild the table views with the updated ones
  table_view const left  = scatter_columns(matched.second.front(), left_on, left_input);
  table_view const right = scatter_columns(matched.second.back(), right_on, right_input);

  auto join_indices = left_join(left.select(left_on), right.select(right_on), compare_nulls);

  if ((left_on.empty() || right_on.empty()) ||
      is_trivial_join(left, right, cudf::detail::join_kind::LEFT_JOIN)) {
    auto probe_build_pair = get_empty_joined_table(left, right);
    return cudf::detail::combine_table_pair(std::move(probe_build_pair.first),
                                            std::move(probe_build_pair.second));
  }
  std::unique_ptr<table> left_result  = detail::gather(left,
                                                      join_indices.first->begin(),
                                                      join_indices.first->end(),
                                                      out_of_bounds_policy::NULLIFY,
                                                      stream,
                                                      mr);
  std::unique_ptr<table> right_result = detail::gather(right,
                                                       join_indices.second->begin(),
                                                       join_indices.second->end(),
                                                       out_of_bounds_policy::NULLIFY,
                                                       stream,
                                                       mr);
  return combine_table_pair(std::move(left_result), std::move(right_result));
}

std::pair<std::unique_ptr<rmm::device_uvector<size_type>>,
          std::unique_ptr<rmm::device_uvector<size_type>>>
full_join(table_view const& left_input,
          table_view const& right_input,
          null_equality compare_nulls,
          rmm::cuda_stream_view stream,
          rmm::mr::device_memory_resource* mr)
{
  // Make sure any dictionary columns have matched key sets.
  // This will return any new dictionary columns created as well as updated table_views.
  auto matched = cudf::dictionary::detail::match_dictionaries(
    {left_input, right_input},  // these should match
    stream,
    rmm::mr::get_current_device_resource());  // temporary objects returned
  // now rebuild the table views with the updated ones
  table_view const left  = matched.second.front();
  table_view const right = matched.second.back();

  cudf::hash_join hj_obj(right, compare_nulls, stream);
  return hj_obj.full_join(left, compare_nulls, std::nullopt, stream, mr);
}

std::unique_ptr<table> full_join(table_view const& left_input,
                                 table_view const& right_input,
                                 std::vector<size_type> const& left_on,
                                 std::vector<size_type> const& right_on,
                                 null_equality compare_nulls,
                                 rmm::cuda_stream_view stream,
                                 rmm::mr::device_memory_resource* mr)
{
  // Make sure any dictionary columns have matched key sets.
  // This will return any new dictionary columns created as well as updated table_views.
  auto matched = cudf::dictionary::detail::match_dictionaries(
    {left_input.select(left_on), right_input.select(right_on)},  // these should match
    stream,
    rmm::mr::get_current_device_resource());  // temporary objects returned
  // now rebuild the table views with the updated ones
  table_view const left  = scatter_columns(matched.second.front(), left_on, left_input);
  table_view const right = scatter_columns(matched.second.back(), right_on, right_input);

  auto join_indices = full_join(left.select(left_on), right.select(right_on), compare_nulls);

  if ((left_on.empty() || right_on.empty()) ||
      is_trivial_join(left, right, cudf::detail::join_kind::FULL_JOIN)) {
    auto probe_build_pair = get_empty_joined_table(left, right);
    return cudf::detail::combine_table_pair(std::move(probe_build_pair.first),
                                            std::move(probe_build_pair.second));
  }
  std::unique_ptr<table> left_result  = detail::gather(left,
                                                      join_indices.first->begin(),
                                                      join_indices.first->end(),
                                                      out_of_bounds_policy::NULLIFY,
                                                      stream,
                                                      mr);
  std::unique_ptr<table> right_result = detail::gather(right,
                                                       join_indices.second->begin(),
                                                       join_indices.second->end(),
                                                       out_of_bounds_policy::NULLIFY,
                                                       stream,
                                                       mr);
  return combine_table_pair(std::move(left_result), std::move(right_result));
}

}  // namespace detail

hash_join::~hash_join() = default;

hash_join::hash_join(cudf::table_view const& build,
                     null_equality compare_nulls,
                     rmm::cuda_stream_view stream)
  : impl{std::make_unique<const hash_join::hash_join_impl>(build, compare_nulls, stream)}
{
}

std::pair<std::unique_ptr<rmm::device_uvector<size_type>>,
          std::unique_ptr<rmm::device_uvector<size_type>>>
hash_join::inner_join(cudf::table_view const& probe,
                      null_equality compare_nulls,
                      std::optional<std::size_t> output_size,
                      rmm::cuda_stream_view stream,
                      rmm::mr::device_memory_resource* mr) const
{
  return impl->inner_join(probe, compare_nulls, output_size, stream, mr);
}

std::pair<std::unique_ptr<rmm::device_uvector<size_type>>,
          std::unique_ptr<rmm::device_uvector<size_type>>>
hash_join::left_join(cudf::table_view const& probe,
                     null_equality compare_nulls,
                     std::optional<std::size_t> output_size,
                     rmm::cuda_stream_view stream,
                     rmm::mr::device_memory_resource* mr) const
{
  return impl->left_join(probe, compare_nulls, output_size, stream, mr);
}

std::pair<std::unique_ptr<rmm::device_uvector<size_type>>,
          std::unique_ptr<rmm::device_uvector<size_type>>>
hash_join::full_join(cudf::table_view const& probe,
                     null_equality compare_nulls,
                     std::optional<std::size_t> output_size,
                     rmm::cuda_stream_view stream,
                     rmm::mr::device_memory_resource* mr) const
{
  return impl->full_join(probe, compare_nulls, output_size, stream, mr);
}

std::size_t hash_join::inner_join_size(cudf::table_view const& probe,
                                       null_equality compare_nulls,
                                       rmm::cuda_stream_view stream) const
{
  return impl->inner_join_size(probe, compare_nulls, stream);
}

std::size_t hash_join::left_join_size(cudf::table_view const& probe,
                                      null_equality compare_nulls,
                                      rmm::cuda_stream_view stream) const
{
  return impl->left_join_size(probe, compare_nulls, stream);
}

std::size_t hash_join::full_join_size(cudf::table_view const& probe,
                                      null_equality compare_nulls,
                                      rmm::cuda_stream_view stream,
                                      rmm::mr::device_memory_resource* mr) const
{
  return impl->full_join_size(probe, compare_nulls, stream, mr);
}

// external APIs

std::pair<std::unique_ptr<rmm::device_uvector<size_type>>,
          std::unique_ptr<rmm::device_uvector<size_type>>>
inner_join(table_view const& left,
           table_view const& right,
           null_equality compare_nulls,
           rmm::mr::device_memory_resource* mr)
{
  CUDF_FUNC_RANGE();
  return detail::inner_join(left, right, compare_nulls, rmm::cuda_stream_default, mr);
}

std::unique_ptr<table> inner_join(table_view const& left,
                                  table_view const& right,
                                  std::vector<size_type> const& left_on,
                                  std::vector<size_type> const& right_on,
                                  null_equality compare_nulls,
                                  rmm::mr::device_memory_resource* mr)
{
  CUDF_FUNC_RANGE();
  return detail::inner_join(
    left, right, left_on, right_on, compare_nulls, rmm::cuda_stream_default, mr);
}

std::pair<std::unique_ptr<rmm::device_uvector<size_type>>,
          std::unique_ptr<rmm::device_uvector<size_type>>>
left_join(table_view const& left,
          table_view const& right,
          null_equality compare_nulls,
          rmm::mr::device_memory_resource* mr)
{
  CUDF_FUNC_RANGE();
  return detail::left_join(left, right, compare_nulls, rmm::cuda_stream_default, mr);
}

std::unique_ptr<table> left_join(table_view const& left,
                                 table_view const& right,
                                 std::vector<size_type> const& left_on,
                                 std::vector<size_type> const& right_on,
                                 null_equality compare_nulls,
                                 rmm::mr::device_memory_resource* mr)
{
  CUDF_FUNC_RANGE();
  return detail::left_join(
    left, right, left_on, right_on, compare_nulls, rmm::cuda_stream_default, mr);
}

std::pair<std::unique_ptr<rmm::device_uvector<size_type>>,
          std::unique_ptr<rmm::device_uvector<size_type>>>
full_join(table_view const& left,
          table_view const& right,
          null_equality compare_nulls,
          rmm::mr::device_memory_resource* mr)
{
  CUDF_FUNC_RANGE();
  return detail::full_join(left, right, compare_nulls, rmm::cuda_stream_default, mr);
}

std::unique_ptr<table> full_join(table_view const& left,
                                 table_view const& right,
                                 std::vector<size_type> const& left_on,
                                 std::vector<size_type> const& right_on,
                                 null_equality compare_nulls,
                                 rmm::mr::device_memory_resource* mr)
{
  CUDF_FUNC_RANGE();
  return detail::full_join(
    left, right, left_on, right_on, compare_nulls, rmm::cuda_stream_default, mr);
}

}  // namespace cudf
