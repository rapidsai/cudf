/*
 * Copyright (c) 2019-2020, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */
#include <cudf/copying.hpp>
#include <cudf/detail/concatenate.cuh>
#include <cudf/detail/gather.cuh>
#include <cudf/detail/gather.hpp>
#include <cudf/detail/nvtx/ranges.hpp>
#include <cudf/join.hpp>
#include <cudf/table/table.hpp>
#include <cudf/table/table_view.hpp>
#include <cudf/utilities/error.hpp>

#include "cudf/detail/utilities/cuda.cuh"
#include "cudf/types.hpp"
#include "hash_join.cuh"
#include "join_common_utils.hpp"
#include "nested_loop_join.cuh"

namespace cudf {
namespace detail {

/**
 * @brief Returns a vector with non-common indices which is set difference
 * between `[0, num_columns)` and index values in common_column_indices
 *
 * @param num_columns The number of columns, which represents column indices
 * from `[0, num_columns)` in a table
 * @param common_column_indices A vector of common indices which needs to be
 * excluded from `[0, num_columns)`
 *
 * @return vector A vector containing only the indices which are not present in
 * `common_column_indices`
 */
auto non_common_column_indices(size_type num_columns,
                               std::vector<size_type> const& common_column_indices)
{
  CUDF_EXPECTS(common_column_indices.size() <= static_cast<unsigned long>(num_columns),
               "Too many columns in common");
  std::vector<size_type> all_column_indices(num_columns);
  std::iota(std::begin(all_column_indices), std::end(all_column_indices), 0);
  std::vector<size_type> sorted_common_column_indices{common_column_indices};
  std::sort(std::begin(sorted_common_column_indices), std::end(sorted_common_column_indices));
  std::vector<size_type> non_common_column_indices(num_columns - common_column_indices.size());
  std::set_difference(std::cbegin(all_column_indices),
                      std::cend(all_column_indices),
                      std::cbegin(sorted_common_column_indices),
                      std::cend(sorted_common_column_indices),
                      std::begin(non_common_column_indices));
  return non_common_column_indices;
}

std::unique_ptr<table> get_empty_joined_table(
  table_view const& left,
  table_view const& right,
  std::vector<std::pair<size_type, size_type>> const& columns_in_common)
{
  std::vector<size_type> right_columns_in_common(columns_in_common.size());
  std::transform(columns_in_common.begin(),
                 columns_in_common.end(),
                 right_columns_in_common.begin(),
                 [](auto& col) { return col.second; });
  std::unique_ptr<table> empty_left  = empty_like(left);
  std::unique_ptr<table> empty_right = empty_like(right);
  std::vector<size_type> right_non_common_indices =
    non_common_column_indices(right.num_columns(), right_columns_in_common);
  table_view tmp_right_table = (*empty_right).select(right_non_common_indices);
  table_view tmp_table{{*empty_left, tmp_right_table}};
  return std::make_unique<table>(tmp_table);
}

VectorPair concatenate_vector_pairs(VectorPair& a, VectorPair& b)
{
  CUDF_EXPECTS((a.first.size() == a.second.size()),
               "Mismatch between sizes of vectors in vector pair");
  CUDF_EXPECTS((b.first.size() == b.second.size()),
               "Mismatch between sizes of vectors in vector pair");
  if (a.first.size() == 0) {
    return b;
  } else if (b.first.size() == 0) {
    return a;
  }
  auto original_size = a.first.size();
  a.first.resize(a.first.size() + b.first.size());
  a.second.resize(a.second.size() + b.second.size());
  thrust::copy(b.first.begin(), b.first.end(), a.first.begin() + original_size);
  thrust::copy(b.second.begin(), b.second.end(), a.second.begin() + original_size);
  return a;
}

template <typename T>
struct valid_range {
  T start, stop;
  __host__ __device__ valid_range(const T begin, const T end) : start(begin), stop(end) {}

  __host__ __device__ __forceinline__ bool operator()(const T index)
  {
    return ((index >= start) && (index < stop));
  }
};

/**
 * @brief  Creates a table containing the complement of left join indices.
 * This table has two columns. The first one is filled with JoinNoneValue(-1)
 * and the second one contains values from 0 to right_table_row_count - 1
 * excluding those found in the right_indices column.
 *
 * @param right_indices Vector of indices
 * @param left_table_row_count Number of rows of left table
 * @param right_table_row_count Number of rows of right table
 * @param stream CUDA stream used for device memory operations and kernel launches.
 *
 * @return Pair of vectors containing the left join indices complement
 */
std::pair<rmm::device_vector<size_type>, rmm::device_vector<size_type>>
get_left_join_indices_complement(rmm::device_vector<size_type>& right_indices,
                                 size_type left_table_row_count,
                                 size_type right_table_row_count,
                                 hipStream_t stream)
{
  // Get array of indices that do not appear in right_indices

  // Vector allocated for unmatched result
  rmm::device_vector<size_type> right_indices_complement(right_table_row_count);

  // If left table is empty in a full join call then all rows of the right table
  // should be represented in the joined indices. This is an optimization since
  // if left table is empty and full join is called all the elements in
  // right_indices will be JoinNoneValue, i.e. -1. This if path should
  // produce exactly the same result as the else path but will be faster.
  if (left_table_row_count == 0) {
    thrust::sequence(rmm::exec_policy(stream)->on(stream),
                     right_indices_complement.begin(),
                     right_indices_complement.end(),
                     0);
  } else {
    // Assume all the indices in invalid_index_map are invalid
    rmm::device_vector<size_type> invalid_index_map(right_table_row_count, 1);
    // Functor to check for index validity since left joins can create invalid indices
    valid_range<size_type> valid(0, right_table_row_count);

    // invalid_index_map[index_ptr[i]] = 0 for i = 0 to right_table_row_count
    // Thus specifying that those locations are valid
    thrust::scatter_if(rmm::exec_policy(stream)->on(stream),
                       thrust::make_constant_iterator(0),
                       thrust::make_constant_iterator(0) + right_indices.size(),
                       right_indices.begin(),      // Index locations
                       right_indices.begin(),      // Stencil - Check if index location is valid
                       invalid_index_map.begin(),  // Output indices
                       valid);                     // Stencil Predicate
    size_type begin_counter = static_cast<size_type>(0);
    size_type end_counter   = static_cast<size_type>(right_table_row_count);

    // Create list of indices that have been marked as invalid
    size_type indices_count = thrust::copy_if(rmm::exec_policy(stream)->on(stream),
                                              thrust::make_counting_iterator(begin_counter),
                                              thrust::make_counting_iterator(end_counter),
                                              invalid_index_map.begin(),
                                              right_indices_complement.begin(),
                                              thrust::identity<size_type>()) -
                              right_indices_complement.begin();
    right_indices_complement.resize(indices_count);
  }

  rmm::device_vector<size_type> left_invalid_indices(right_indices_complement.size(),
                                                     JoinNoneValue);

  return std::make_pair(std::move(left_invalid_indices), std::move(right_indices_complement));
}

/**
 * @brief Computes the base join operation between two tables and returns the
 * output indices of left and right table as a combined table, i.e. if full
 * join is specified as the join type then left join is called.
 *
 * @throw cudf::logic_error if `left` or `right` table is empty
 * @throw cudf::logic_error if types do not match between joining columns
 *
 * @tparam JoinKind The type of join to be performed
 *
 * @param left  Table of left columns to join
 * @param right Table of right  columns to join
 * @param compare_nulls Controls whether null join-key values should match or not.
 * @param stream CUDA stream used for device memory operations and kernel launches.
 *
 * @return Join output indices vector pair
 */
template <join_kind JoinKind>
std::pair<rmm::device_vector<size_type>, rmm::device_vector<size_type>> get_base_join_indices(
  table_view const& left, table_view const& right, null_equality compare_nulls, hipStream_t stream)
{
  CUDF_EXPECTS(0 != left.num_columns(), "Selected left dataset is empty");
  CUDF_EXPECTS(0 != right.num_columns(), "Selected right dataset is empty");
  CUDF_EXPECTS(std::equal(std::cbegin(left),
                          std::cend(left),
                          std::cbegin(right),
                          std::cend(right),
                          [](const auto& l, const auto& r) { return l.type() == r.type(); }),
               "Mismatch in joining column data types");

  constexpr join_kind BaseJoinKind =
    (JoinKind == join_kind::FULL_JOIN) ? join_kind::LEFT_JOIN : JoinKind;
  return get_base_hash_join_indices<BaseJoinKind>(left, right, false, compare_nulls, stream);
}

/**
 * @brief  Combines the non common left, common left and non common right
 * columns in the correct order to form the join output table.
 *
 * @param left_noncommon_cols Columns obtained by gathering non common left
 * columns.
 * @param left_noncommon_col_indices Output locations of non common left columns
 * in the final table output
 * @param left_common_cols Columns obtained by gathering common left
 * columns.
 * @param left_common_col_indices Output locations of common left columns in the
 * final table output
 * @param right_noncommon_cols Table obtained by gathering non common right
 * columns.
 *
 * @return Rearranged columns.
 */
std::vector<std::unique_ptr<column>> combine_join_columns(
  std::vector<std::unique_ptr<column>>&& left_noncommon_cols,
  std::vector<size_type> const& left_noncommon_col_indices,
  std::vector<std::unique_ptr<column>>&& left_common_cols,
  std::vector<size_type> const& left_common_col_indices,
  std::vector<std::unique_ptr<column>>&& right_noncommon_cols)
{
  std::vector<std::unique_ptr<column>> combined_cols(left_noncommon_cols.size() +
                                                     left_common_cols.size());
  for (size_t i = 0; i < left_noncommon_cols.size(); ++i) {
    combined_cols.at(left_noncommon_col_indices.at(i)) = std::move(left_noncommon_cols.at(i));
  }
  for (size_t i = 0; i < left_common_cols.size(); ++i) {
    combined_cols.at(left_common_col_indices.at(i)) = std::move(left_common_cols.at(i));
  }
  combined_cols.insert(combined_cols.end(),
                       std::make_move_iterator(right_noncommon_cols.begin()),
                       std::make_move_iterator(right_noncommon_cols.end()));
  return combined_cols;
}

/**
 * @brief  Gathers rows from `left` and `right` table and combines them into a
 * single table.
 *
 * @tparam JoinKind The type of join to be performed
 *
 * @param left Left input table
 * @param right Right input table
 * @param joined_indices Pair of vectors containing row indices from which
 * `left` and `right` tables are gathered. If any row index is out of bounds,
 * the contribution in the output `table` will be NULL.
 * @param columns_in_common is a vector of pairs of column indices
 * from tables `left` and `right` respectively, that are "in common".
 * For "common" columns, only a single output column will be produced.
 * For an inner or left join, the result will be gathered from the column in
 * `left`. For a full join, the result will be gathered from both common
 * columns in `left` and `right` and concatenated to form a single column.
 *
 * @return `table` containing the concatenation of rows from `left` and
 * `right` specified by `joined_indices`.
 * For any columns indicated by `columns_in_common`, only the corresponding
 * column in `left` will be included in the result. Final form would look like
 * `left(including common columns)+right(excluding common columns)`.
 */
template <join_kind JoinKind>
std::unique_ptr<table> construct_join_output_df(
  table_view const& left,
  table_view const& right,
  VectorPair& joined_indices,
  std::vector<std::pair<size_type, size_type>> const& columns_in_common,
  rmm::mr::device_memory_resource* mr,
  hipStream_t stream)
{
  std::vector<size_type> left_common_col;
  left_common_col.reserve(columns_in_common.size());
  std::vector<size_type> right_common_col;
  right_common_col.reserve(columns_in_common.size());
  for (const auto& c : columns_in_common) {
    left_common_col.push_back(c.first);
    right_common_col.push_back(c.second);
  }
  std::vector<size_type> left_noncommon_col =
    non_common_column_indices(left.num_columns(), left_common_col);
  std::vector<size_type> right_noncommon_col =
    non_common_column_indices(right.num_columns(), right_common_col);

  bool const nullify_out_of_bounds{JoinKind != join_kind::INNER_JOIN};

  std::unique_ptr<table> common_table = std::make_unique<table>();
  // Construct the joined columns
  if (join_kind::FULL_JOIN == JoinKind) {
    auto complement_indices = get_left_join_indices_complement(
      joined_indices.second, left.num_rows(), right.num_rows(), stream);
    if (not columns_in_common.empty()) {
      auto common_from_right = detail::gather(right.select(right_common_col),
                                              complement_indices.second.begin(),
                                              complement_indices.second.end(),
                                              nullify_out_of_bounds,
                                              rmm::mr::get_default_resource(),
                                              stream);
      auto common_from_left  = detail::gather(left.select(left_common_col),
                                             joined_indices.first.begin(),
                                             joined_indices.first.end(),
                                             nullify_out_of_bounds,
                                             rmm::mr::get_default_resource(),
                                             stream);
      common_table           = cudf::detail::concatenate(
        {common_from_right->view(), common_from_left->view()}, mr, stream);
    }
    joined_indices = concatenate_vector_pairs(complement_indices, joined_indices);
  } else {
    if (not columns_in_common.empty()) {
      common_table = detail::gather(left.select(left_common_col),
                                    joined_indices.first.begin(),
                                    joined_indices.first.end(),
                                    nullify_out_of_bounds,
                                    mr,
                                    stream);
    }
  }

  // Construct the left non common columns
  std::unique_ptr<table> left_table = detail::gather(left.select(left_noncommon_col),
                                                     joined_indices.first.begin(),
                                                     joined_indices.first.end(),
                                                     nullify_out_of_bounds,
                                                     mr,
                                                     stream);

  std::unique_ptr<table> right_table = detail::gather(right.select(right_noncommon_col),
                                                      joined_indices.second.begin(),
                                                      joined_indices.second.end(),
                                                      nullify_out_of_bounds,
                                                      mr,
                                                      stream);

  return std::make_unique<table>(combine_join_columns(left_table->release(),
                                                      left_noncommon_col,
                                                      common_table->release(),
                                                      left_common_col,
                                                      right_table->release()));
}

/**
 * @brief  Performs join on the columns provided in `left` and `right` as per
 * the joining indices given in `left_on` and `right_on` and creates a single
 * table.
 *
 * @throw cudf::logic_error if `columns_in_common` contains a pair of indices
 * (`L`, `R`) where `L` does not exist in `left_on` or `R` does not exist in
 * `right_on`.
 * @throw cudf::logic_error if `columns_in_common` contains a pair of indices
 * (`L`, `R`) such that the location of `L` within `left_on` is not equal to
 * the location of `R` within `right_on`.
 * @throw cudf::logic_error if the number of elements in `left_on` or
 * `right_on` are not equal.
 * @throw cudf::logic_error if the number of columns in either `left` or
 * `right` table is 0 or exceeds MAX_JOIN_SIZE.
 * @throw std::out_of_range if elements of `left_on` or `right_on` exceed the
 * number of columns in the left or right table.
 *
 * @tparam JoinKind The type of join to be performed
 *
 * @param left The left table
 * @param right The right table
 * @param left_on The column's indices from `left` to join on.
 * Column `i` from `left_on` will be compared against column `i` of `right_on`.
 * @param right_on The column's indices from `right` to join on.
 * Column `i` from `right_on` will be compared with column `i` of `left_on`.
 * @param columns_in_common is a vector of pairs of column indices into
 * `left_on` and `right_on`, respectively, that are "in common". For "common"
 * columns, only a single output column will be produced, which is gathered
 * from `left_on` if it is left join or from intersection of `left_on` and
 * `right_on` if it is inner join or gathered from both `left_on` and
 * `right_on` if it is full join. Else, for every column in `left_on` and
 * `right_on`, an output column will be produced.
 * @param compare_nulls Controls whether null join-key values should match or not.
 * @param mr Device memory resource used to allocate the returned table's device memory
 * @param stream CUDA stream used for device memory operations and kernel launches.
 *
 * @return Result of joining `left` and `right` tables on the columns specified
 * by `left_on` and `right_on`. The resulting table will be joined columns of
 * `left(including common columns)+right(excluding common columns)`.
 */
template <join_kind JoinKind>
std::unique_ptr<table> join_call_compute_df(
  table_view const& left,
  table_view const& right,
  std::vector<size_type> const& left_on,
  std::vector<size_type> const& right_on,
  std::vector<std::pair<size_type, size_type>> const& columns_in_common,
  null_equality compare_nulls,
  rmm::mr::device_memory_resource* mr,
  hipStream_t stream = 0)
{
  CUDF_EXPECTS(0 != left.num_columns(), "Left table is empty");
  CUDF_EXPECTS(0 != right.num_columns(), "Right table is empty");
  CUDF_EXPECTS(left.num_rows() < MAX_JOIN_SIZE, "Left column size is too big");
  CUDF_EXPECTS(right.num_rows() < MAX_JOIN_SIZE, "Right column size is too big");

  CUDF_EXPECTS(left_on.size() == right_on.size(), "Mismatch in number of columns to be joined on");

  CUDF_EXPECTS(std::all_of(columns_in_common.begin(),
                           columns_in_common.end(),
                           [&left_on, &right_on](auto p) {
                             size_t lind =
                               std::find(left_on.begin(), left_on.end(), p.first) - left_on.begin();
                             size_t rind = std::find(right_on.begin(), right_on.end(), p.second) -
                                           right_on.begin();
                             return (lind != left_on.size()) && (rind != right_on.size()) &&
                                    (lind == rind);
                           }),
               "Invalid values passed to columns_in_common");

  if (is_trivial_join(left, right, left_on, right_on, JoinKind)) {
    return get_empty_joined_table(left, right, columns_in_common);
  }

  auto joined_indices = get_base_join_indices<JoinKind>(
    left.select(left_on), right.select(right_on), compare_nulls, stream);

  return construct_join_output_df<JoinKind>(
    left, right, joined_indices, columns_in_common, mr, stream);
}

}  // namespace detail

std::unique_ptr<table> inner_join(
  table_view const& left,
  table_view const& right,
  std::vector<size_type> const& left_on,
  std::vector<size_type> const& right_on,
  std::vector<std::pair<size_type, size_type>> const& columns_in_common,
  null_equality compare_nulls,
  rmm::mr::device_memory_resource* mr)
{
  CUDF_FUNC_RANGE();
  return detail::join_call_compute_df<::cudf::detail::join_kind::INNER_JOIN>(
    left, right, left_on, right_on, columns_in_common, compare_nulls, mr);
}

std::unique_ptr<table> left_join(
  table_view const& left,
  table_view const& right,
  std::vector<size_type> const& left_on,
  std::vector<size_type> const& right_on,
  std::vector<std::pair<size_type, size_type>> const& columns_in_common,
  null_equality compare_nulls,
  rmm::mr::device_memory_resource* mr)
{
  CUDF_FUNC_RANGE();
  return detail::join_call_compute_df<::cudf::detail::join_kind::LEFT_JOIN>(
    left, right, left_on, right_on, columns_in_common, compare_nulls, mr);
}

std::unique_ptr<table> full_join(
  table_view const& left,
  table_view const& right,
  std::vector<size_type> const& left_on,
  std::vector<size_type> const& right_on,
  std::vector<std::pair<size_type, size_type>> const& columns_in_common,
  null_equality compare_nulls,
  rmm::mr::device_memory_resource* mr)
{
  CUDF_FUNC_RANGE();
  return detail::join_call_compute_df<::cudf::detail::join_kind::FULL_JOIN>(
    left, right, left_on, right_on, columns_in_common, compare_nulls, mr);
}

}  // namespace cudf
