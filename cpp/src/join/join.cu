/*
 * Copyright (c) 2019, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */
#include <cudf/copying.hpp>
#include <cudf/join.hpp>
#include <cudf/table/table_view.hpp>
#include <cudf/table/table.hpp>
#include <cudf/utilities/error.hpp>
#include <cudf/concatenate.hpp>
#include <cudf/detail/gather.hpp>
#include <cudf/detail/gather.cuh>
#include <cudf/detail/nvtx/ranges.hpp>

#include <join/join_common_utils.hpp>
#include <join/hash_join.cuh>

namespace cudf {

namespace experimental {

namespace detail {

/**---------------------------------------------------------------------------*
 * @brief Returns a vector with non-common indices which is set difference
 * between `[0, num_columns)` and index values in common_column_indices
 *
 * @param num_columns The number of columns , which represents column indices
 * from `[0, num_columns)` in a table
 * @param common_column_indices A vector of common indices which needs to be
 * excluded from `[0, num_columns)`
 * @return vector A vector containing only the indices which are not present in
 * `common_column_indices`
 *---------------------------------------------------------------------------**/
auto non_common_column_indices(
    size_type num_columns,
    std::vector<size_type> const& common_column_indices) {
  CUDF_EXPECTS(common_column_indices.size() <= static_cast<unsigned long>(num_columns),
               "Too many columns in common");
  std::vector<size_type> all_column_indices(num_columns);
  std::iota(std::begin(all_column_indices), std::end(all_column_indices), 0);
  std::vector<size_type> sorted_common_column_indices{
      common_column_indices};
  std::sort(std::begin(sorted_common_column_indices),
            std::end(sorted_common_column_indices));
  std::vector<size_type> non_common_column_indices(num_columns -
                                                common_column_indices.size());
  std::set_difference(std::cbegin(all_column_indices),
                      std::cend(all_column_indices),
                      std::cbegin(sorted_common_column_indices),
                      std::cend(sorted_common_column_indices), std::begin(non_common_column_indices));
   return non_common_column_indices;
}


std::unique_ptr<experimental::table> get_empty_joined_table(
                         table_view const& left,
                         table_view const& right,
                         std::vector<std::pair<size_type, size_type>> const& columns_in_common) {
  std::vector<size_type> right_columns_in_common (columns_in_common.size());
  std::transform(
      columns_in_common.begin(),
      columns_in_common.end(),
      right_columns_in_common.begin(),
      [](auto& col) { return col.second; } );
  std::unique_ptr<experimental::table> empty_left = experimental::empty_like(left);
  std::unique_ptr<experimental::table> empty_right = experimental::empty_like(right);
  std::vector <size_type> right_non_common_indices =
    non_common_column_indices(right.num_columns(), right_columns_in_common);
  table_view tmp_right_table = (*empty_right).select(right_non_common_indices);
  table_view tmp_table{{*empty_left, tmp_right_table}};
  return std::make_unique<experimental::table>(tmp_table);
}


VectorPair
concatenate_vector_pairs(
  VectorPair& a, VectorPair& b)
{
  CUDF_EXPECTS((a.first.size() == a.second.size()),
               "Mismatch between sizes of vectors in vector pair");
  CUDF_EXPECTS((b.first.size() == b.second.size()),
               "Mismatch between sizes of vectors in vector pair");
  if (a.first.size() == 0) {
    return b;
  } else if (b.first.size() == 0) {
    return a;
  }
  auto original_size = a.first.size();
  a.first.resize(a.first.size() + b.first.size());
  a.second.resize(a.second.size() + b.second.size());
  thrust::copy(b.first.begin(), b.first.end(), a.first.begin() + original_size);
  thrust::copy(b.second.begin(), b.second.end(), a.second.begin() + original_size);
  return a;
}


template <typename T>
struct valid_range {
    T start, stop;
    __host__ __device__
    valid_range(
            const T begin,
            const T end) :
        start(begin), stop(end) {}

    __host__ __device__ __forceinline__
    bool operator()(const T index)
    {
        return ((index >= start) && (index < stop));
    }
};


/* --------------------------------------------------------------------------*/
/**
* @brief  Creates a table containing the complement of left join indices.
* This table has two columns. The first one is filled with JoinNoneValue(-1)
* and the second one contains values from 0 to right_table_row_count - 1
* excluding those found in the right_indices column.
*
* @Param right_indices Vector of indices
* @Param left_table_row_count Number of rows of left table
* @Param right_table_row_count Number of rows of right table
* @param stream Optional, stream on which all memory allocations and copies
* will be performed
*
* @Returns  Pair of vectors containing the left join indices complement
*/
/* ----------------------------------------------------------------------------*/
std::pair<rmm::device_vector<size_type>,
rmm::device_vector<size_type>>
get_left_join_indices_complement(
    rmm::device_vector<size_type>& right_indices,
    size_type left_table_row_count,
    size_type right_table_row_count,
    hipStream_t stream) {

  //Get array of indices that do not appear in right_indices

  //Vector allocated for unmatched result
  rmm::device_vector<size_type> right_indices_complement(right_table_row_count);

  //If left table is empty in a full join call then all rows of the right table
  //should be represented in the joined indices. This is an optimization since
  //if left table is empty and full join is called all the elements in
  //right_indices will be JoinNoneValue, i.e. -1. This if path should
  //produce exactly the same result as the else path but will be faster.
  if (left_table_row_count == 0) {
    thrust::sequence(
        rmm::exec_policy(stream)->on(stream),
        right_indices_complement.begin(),
        right_indices_complement.end(),
        0);
  } else {
    //Assume all the indices in invalid_index_map are invalid
    rmm::device_vector<size_type> invalid_index_map(right_table_row_count, 1);
    //Functor to check for index validity since left joins can create invalid indices
    valid_range<size_type> valid(0, right_table_row_count);

    //invalid_index_map[index_ptr[i]] = 0 for i = 0 to right_table_row_count
    //Thus specifying that those locations are valid
    thrust::scatter_if(
        rmm::exec_policy(stream)->on(stream),
        thrust::make_constant_iterator(0),
        thrust::make_constant_iterator(0) + right_indices.size(),
        right_indices.begin(),//Index locations
        right_indices.begin(),//Stencil - Check if index location is valid
        invalid_index_map.begin(),//Output indices
        valid);//Stencil Predicate
    size_type begin_counter = static_cast<size_type>(0);
    size_type end_counter = static_cast<size_type>(right_table_row_count);

    //Create list of indices that have been marked as invalid
    size_type indices_count = thrust::copy_if(
        rmm::exec_policy(stream)->on(stream),
        thrust::make_counting_iterator(begin_counter),
        thrust::make_counting_iterator(end_counter),
        invalid_index_map.begin(),
        right_indices_complement.begin(),
        thrust::identity<size_type>()) -
      right_indices_complement.begin();
    right_indices_complement.resize(indices_count);
  }

  rmm::device_vector<size_type> left_invalid_indices(
      right_indices_complement.size(), JoinNoneValue);

  return std::make_pair(std::move(left_invalid_indices), std::move(right_indices_complement));
}


/* --------------------------------------------------------------------------*/
/**
 * @brief  Computes the base join operation between two tables and returns the
 * output indices of left and right table as a combined table, i.e. if full
 * join is specified as the join type then left join is called.
 *
 * @throws cudf::logic_error
 * If `left`/`right` table is empty
 * If type mismatch between joining columns
 *
 * @param left  Table of left columns to join
 * @param right Table of right  columns to join
 * @param stream stream on which all memory allocations and copies
 * will be performed
 * @tparam join_kind The type of join to be performed
 *
 * @returns Join output indices vector pair
 */
/* ----------------------------------------------------------------------------*/
template <join_kind JoinKind>
std::pair<rmm::device_vector<size_type>,
  rmm::device_vector<size_type>>
get_base_join_indices(
    table_view const& left,
    table_view const& right,
    hipStream_t stream) {
  CUDF_EXPECTS (0 != left.num_columns(), "Selected left dataset is empty");
  CUDF_EXPECTS (0 != right.num_columns(), "Selected right dataset is empty");
  CUDF_EXPECTS(std::equal(
      std::cbegin(left), std::cend(left),
      std::cbegin(right), std::cend(right),
      [](const auto &l, const auto &r) {
      return l.type() == r.type(); }),
      "Mismatch in joining column data types");

  constexpr join_kind BaseJoinKind = (JoinKind == join_kind::FULL_JOIN)? join_kind::LEFT_JOIN : JoinKind;
  return get_base_hash_join_indices<BaseJoinKind>(left, right, false, stream);
}

/* --------------------------------------------------------------------------*/
/**
* @brief  Combines the non common left, common left and non common right
* columns in the correct order to form the join output table.
*
* @param left_noncommon_cols Columns obtained by gathering non common left
* columns.
* @param left_noncommon_col_indices Output locations of non common left columns
* in the final table output
* @param left_common_cols Columns obtained by gathering common left
* columns.
* @param left_common_col_indices Output locations of common left columns in the
* final table output
* @param right_noncommon_cols Table obtained by gathering non common right
* columns.
*
* @Returns  Table containing rearranged columns.
*/
/* ----------------------------------------------------------------------------*/
std::vector<std::unique_ptr<column>>
combine_join_columns(
    std::vector<std::unique_ptr<column>>&& left_noncommon_cols,
    std::vector<size_type> const& left_noncommon_col_indices,
    std::vector<std::unique_ptr<column>>&& left_common_cols,
    std::vector<size_type> const& left_common_col_indices,
    std::vector<std::unique_ptr<column>>&& right_noncommon_cols) {

  std::vector<std::unique_ptr<column>> combined_cols(
      left_noncommon_cols.size() +
      left_common_cols.size());
  for(size_t i = 0; i < left_noncommon_cols.size(); ++i) {
    combined_cols.at(left_noncommon_col_indices.at(i)) =
      std::move(left_noncommon_cols.at(i));
  }
  for(size_t i = 0; i < left_common_cols.size(); ++i) {
    combined_cols.at(left_common_col_indices.at(i)) = std::move(left_common_cols.at(i));
  }
  combined_cols.insert(combined_cols.end(),
      std::make_move_iterator(right_noncommon_cols.begin()),
      std::make_move_iterator(right_noncommon_cols.end()));
  return combined_cols;
}

/* --------------------------------------------------------------------------*/
/**
* @brief  Gathers rows from `left` and `right` table and combines them into a
* single table.
*
* @param left Left input table
* @param right Right input table
* @param joined_indices Pair of vectors containing row indices from which
* `left` and `right` tables are gathered. If any row index is out of bounds,
* the contribution in the output `table` will be NULL.
* @param columns_in_common is a vector of pairs of column indices
* from tables `left` and `right` respectively, that are "in common".
* For "common" columns, only a single output column will be produced.
* For an inner or left join, the result will be gathered from the column in
* `left`. For a full join, the result will be gathered from both common
* columns in `left` and `right` and concatenated to form a single column.
*
* @Returns `table` containing the concatenation of rows from `left` and
* `right` specified by `joined_indices`.
* For any columns indicated by `columns_in_common`, only the corresponding
* column in `left` will be included in the result. Final form would look like
* `left(including common columns)+right(excluding common columns)`.
*/
/* ----------------------------------------------------------------------------*/
template <join_kind JoinKind>
std::unique_ptr<experimental::table>
construct_join_output_df(
    table_view const& left,
    table_view const& right,
    VectorPair& joined_indices,
    std::vector<std::pair<size_type, size_type>> const& columns_in_common,
    rmm::mr::device_memory_resource* mr,
    hipStream_t stream) {
  std::vector<size_type> left_common_col;
  left_common_col.reserve(columns_in_common.size());
  std::vector<size_type> right_common_col;
  right_common_col.reserve(columns_in_common.size());
  for (const auto c : columns_in_common) {
    left_common_col.push_back(c.first);
    right_common_col.push_back(c.second);
  }
  std::vector<size_type> left_noncommon_col =
    non_common_column_indices(left.num_columns(), left_common_col);
  std::vector<size_type> right_noncommon_col =
    non_common_column_indices(right.num_columns(), right_common_col);

  bool const nullify_out_of_bounds{ JoinKind != join_kind::INNER_JOIN };

  std::unique_ptr<experimental::table> common_table = std::make_unique<experimental::table>();
  // Construct the joined columns
  if (join_kind::FULL_JOIN == JoinKind) {
    auto complement_indices =
      get_left_join_indices_complement(joined_indices.second,
        left.num_rows(), right.num_rows(), stream);
    if (not columns_in_common.empty()) {
      auto common_from_right = experimental::detail::gather(
          right.select(right_common_col),
          complement_indices.second.begin(),
          complement_indices.second.end(),
          nullify_out_of_bounds);
      auto common_from_left = experimental::detail::gather(
          left.select(left_common_col),
          joined_indices.first.begin(),
          joined_indices.first.end(),
          nullify_out_of_bounds);
      common_table = experimental::concatenate(
          {common_from_right->view(), common_from_left->view()});
    } 
    joined_indices =
      concatenate_vector_pairs(complement_indices, joined_indices);
  } else {
      if (not columns_in_common.empty()) {
      common_table = experimental::detail::gather(
          left.select(left_common_col),
          joined_indices.first.begin(),
          joined_indices.first.end(),
          nullify_out_of_bounds);
      }
  }

  // Construct the left non common columns
  std::unique_ptr<experimental::table> left_table =
    experimental::detail::gather(
        left.select(left_noncommon_col),
        joined_indices.first.begin(),
        joined_indices.first.end(),
        nullify_out_of_bounds);

  std::unique_ptr<experimental::table> right_table =
    experimental::detail::gather(
        right.select(right_noncommon_col),
        joined_indices.second.begin(),
        joined_indices.second.end(),
        nullify_out_of_bounds);

  return std::make_unique<experimental::table>(
      combine_join_columns(
      left_table->release(), left_noncommon_col,
      common_table->release(), left_common_col,
      right_table->release()));
}

/* --------------------------------------------------------------------------*/
/**
 * @brief  Performs join on the columns provided in `left` and `right` as per
 * the joining indices given in `left_on` and `right_on` and creates a single
 * table.
 *
 * @throws cudf::logic_error
 * If `columns_in_common` contains a pair of indices (L, R) if L does not exist
 * in `left_on` or R does not exist in `right_on`.
 * If `columns_in_common` contains a pair of indices (L, R) such that the
 * location of `L` within `left_on` is not equal to location of R within
 * `right_on`
 * If number of elements in `left_on` or `right_on` mismatch.
 * If number of columns in either `left` or `right` table is 0 or exceeds
 * MAX_JOIN_SIZE
 * @throws std::out_of_range if element of `left_on` or `right_on` exceed the
 * number of columns in the left or right table.
 *
 * @param left The left table
 * @param right The right table
 * @param left_on The column's indices from `left` to join on.
 * Column `i` from `left_on` will be compared against column `i` of `right_on`.
 * @param right_on The column's indices from `right` to join on.
 * Column `i` from `right_on` will be compared with column `i` of `left_on`.
 * @param columns_in_common is a vector of pairs of column indices into
 * `left_on` and `right_on`, respectively, that are "in common". For "common"
 * columns, only a single output column will be produced, which is gathered
 * from `left_on` if it is left join or from intersection of `left_on` and
 * `right_on`
 * if it is inner join or gathered from both `left_on` and `right_on` if it is
 * full join.
 * Else, for every column in `left_on` and `right_on`, an output column will
 * be produced.
 * @param mr The memory resource that will be used for allocating
 * the device memory for the new table
 * @param stream Optional, stream on which all memory allocations and copies
 * will be performed
 *
 * @tparam join_kind The type of join to be performed
 *
 * @returns Result of joining `left` and `right` tables on the columns
 * specified by `left_on` and `right_on`. The resulting table will be joined columns of
 * `left(including common columns)+right(excluding common columns)`.
 */
/* ----------------------------------------------------------------------------*/
template <join_kind JoinKind>
std::unique_ptr<experimental::table>
join_call_compute_df(
    table_view const& left,
    table_view const& right,
    std::vector<size_type> const& left_on,
    std::vector<size_type> const& right_on,
    std::vector<std::pair<size_type, size_type>> const& columns_in_common,
    rmm::mr::device_memory_resource* mr,
    hipStream_t stream = 0) {

  CUDF_EXPECTS (0 != left.num_columns(), "Left table is empty");
  CUDF_EXPECTS (0 != right.num_columns(), "Right table is empty");
  CUDF_EXPECTS (left.num_rows() < MAX_JOIN_SIZE, "Left column size is too big");
  CUDF_EXPECTS (right.num_rows() < MAX_JOIN_SIZE, "Right column size is too big");

  CUDF_EXPECTS (left_on.size() == right_on.size(), "Mismatch in number of columns to be joined on");

  CUDF_EXPECTS (std::all_of(columns_in_common.begin(), columns_in_common.end(),
      [&left_on, &right_on](auto p){
      size_t lind = std::find(left_on.begin(), left_on.end(), p.first) - left_on.begin();
      size_t rind = std::find(right_on.begin(), right_on.end(), p.second) - right_on.begin();
      return (lind != left_on.size()) && (rind != right_on.size()) && (lind == rind);
      }
      ),
      "Invalid values passed to columns_in_common");

  if (is_trivial_join(left, right, left_on, right_on, JoinKind)) {
    return get_empty_joined_table(left, right, columns_in_common);
  }

  auto joined_indices =
    get_base_join_indices<JoinKind>(left.select(left_on), right.select(right_on), stream);

  return construct_join_output_df<JoinKind>(left, right, joined_indices, columns_in_common, mr, stream);
}

}

std::unique_ptr<experimental::table> inner_join(
                             table_view const& left,
                             table_view const& right,
                             std::vector<size_type> const& left_on,
                             std::vector<size_type> const& right_on,
                             std::vector<std::pair<size_type, size_type>> const& columns_in_common,
                             rmm::mr::device_memory_resource* mr) {
    CUDF_FUNC_RANGE();
    return detail::join_call_compute_df<::cudf::experimental::detail::join_kind::INNER_JOIN>(
        left,
        right,
        left_on,
        right_on,
        columns_in_common,
        mr);
}

std::unique_ptr<experimental::table> left_join(
                             table_view const& left,
                             table_view const& right,
                             std::vector<size_type> const& left_on,
                             std::vector<size_type> const& right_on,
                             std::vector<std::pair<size_type, size_type>> const& columns_in_common,
                             rmm::mr::device_memory_resource* mr) {
    CUDF_FUNC_RANGE();
    return detail::join_call_compute_df<::cudf::experimental::detail::join_kind::LEFT_JOIN>(
           left,
           right,
           left_on,
           right_on,
           columns_in_common,
           mr);
}

std::unique_ptr<experimental::table> full_join(
                             table_view const& left,
                             table_view const& right,
                             std::vector<size_type> const& left_on,
                             std::vector<size_type> const& right_on,
                             std::vector<std::pair<size_type, size_type>> const& columns_in_common,
                         rmm::mr::device_memory_resource* mr) {
    CUDF_FUNC_RANGE();
    return detail::join_call_compute_df<::cudf::experimental::detail::join_kind::FULL_JOIN>(
           left,
           right,
           left_on,
           right_on,
           columns_in_common,
           mr);
}

} //namespace experimental

} //namespace cudf
