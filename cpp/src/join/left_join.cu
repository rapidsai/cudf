#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2025, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "join_common_utils.cuh" 
#include "thrust/iterator/counting_iterator.h"

#include <cudf/detail/cuco_helpers.hpp>
#include <cudf/detail/join/join.hpp>
#include <cudf/detail/join/left_join.cuh>
#include <cudf/detail/null_mask.hpp>
#include <cudf/table/table_view.hpp>

#include <rmm/cuda_stream_view.hpp>
#include <rmm/device_uvector.hpp>

#include <cuco/bucket_storage.cuh>
#include <cuco/detail/open_addressing/kernels.cuh>
#include <cuco/extent.cuh>
#include <cuco/static_set_ref.cuh>
#include <cuco/types.cuh>

#include <algorithm>
#include <limits>

namespace cudf {
namespace detail {
namespace {

auto compute_bucket_storage_size(cudf::table_view tbl, double load_factor)
{
  return std::max({static_cast<cudf::size_type>(
                     cuco::make_valid_extent<left_join::primitive_probing_scheme,
                                             left_join::storage_type,
                                             cudf::size_type>(tbl.num_rows(), load_factor)),
                   static_cast<cudf::size_type>(
                     cuco::make_valid_extent<left_join::nested_probing_scheme,
                                             left_join::storage_type,
                                             cudf::size_type>(tbl.num_rows(), load_factor)),
                   static_cast<cudf::size_type>(
                     cuco::make_valid_extent<left_join::simple_probing_scheme,
                                             left_join::storage_type,
                                             cudf::size_type>(tbl.num_rows(), load_factor))});
}

/**
 * @brief Build a row bitmask for the input table.
 *
 * The output bitmask will have invalid bits corresponding to the input rows having nulls (at
 * any nested level) and vice versa.
 *
 * @param input The input table
 * @param stream CUDA stream used for device memory operations and kernel launches
 * @return A pair of pointer to the output bitmask and the buffer containing the bitmask
 */
std::pair<rmm::device_buffer, bitmask_type const*> build_row_bitmask(table_view const& input,
                                                                     rmm::cuda_stream_view stream)
{
  auto const nullable_columns = get_nullable_columns(input);
  CUDF_EXPECTS(nullable_columns.size() > 0,
               "The input table has nulls thus it should have nullable columns.");

  // If there are more than one nullable column, we compute `bitmask_and` of their null masks.
  // Otherwise, we have only one nullable column and can use its null mask directly.
  if (nullable_columns.size() > 1) {
    auto row_bitmask =
      cudf::detail::bitmask_and(
        table_view{nullable_columns}, stream, cudf::get_current_device_resource_ref())
        .first;
    auto const row_bitmask_ptr = static_cast<bitmask_type const*>(row_bitmask.data());
    return std::pair(std::move(row_bitmask), row_bitmask_ptr);
  }

  return std::pair(rmm::device_buffer{0, stream}, nullable_columns.front().null_mask());
}

}  // namespace

left_join::left_join(cudf::table_view const& build,
                     null_equality compare_nulls,
                     rmm::cuda_stream_view stream)
  // If we cannot know beforehand about null existence then let's assume that there are nulls.
  : left_join(build, compare_nulls, cudf::detail::CUCO_DESIRED_LOAD_FACTOR, stream)
{
}

left_join::left_join(cudf::table_view const& build,
                     null_equality compare_nulls,
                     double load_factor,
                     rmm::cuda_stream_view stream)
  : _has_nested_columns{cudf::has_nested_columns(build)},
    _nulls_equal{compare_nulls},
    _build{build},
    _preprocessed_build{
      cudf::experimental::row::equality::preprocessed_table::create(_build, stream)},
    _bucket_storage{cuco::extent<cudf::size_type>{compute_bucket_storage_size(build, load_factor)},
                    cuco_allocator<char>{rmm::mr::polymorphic_allocator<char>{}, stream.value()}}
{
  auto const build_has_nulls = has_nested_nulls(_build);
  auto const build_has_floating_point =
    std::any_of(_build.begin(), _build.end(), [](auto const& col) {
      return cudf::is_floating_point(col.type());
    });
  auto empty_sentinel_key = cuco::empty_key{key{-1}};
  _bucket_storage.initialize(empty_sentinel_key);

  if (cudf::is_primitive_row_op_compatible(_build) && !build_has_floating_point) {
    auto const d_build_hasher =
      primitive_row_hasher{nullate::DYNAMIC{build_has_nulls}, _preprocessed_build};
    auto const d_build_comparator = cudf::row::primitive::row_equality_comparator{
      nullate::DYNAMIC{build_has_nulls}, _preprocessed_build, _preprocessed_build, compare_nulls};

    cuco::static_set_ref<key, cuda::thread_scope_device, primitive_row_comparator, primitive_probing_scheme, storage_type_ref> set_ref{
      empty_sentinel_key, 
      d_build_comparator, 
      primitive_probing_scheme{d_build_hasher}, 
      cuda::thread_scope_device{}, 
      _bucket_storage.ref()};
      
    // Build hash table by inserting all rows from build table
    auto const grid_size = cuco::detail::grid_size(_build.num_rows(), primitive_probing_scheme::cg_size);

    if (build_has_nulls && compare_nulls == null_equality::UNEQUAL) {
      auto const bitmask_buffer_and_ptr = build_row_bitmask(build, stream);
      auto const row_bitmask_ptr        = bitmask_buffer_and_ptr.second;

      // If the build table has nulls but they are compared unequal, don't insert them.
      // Otherwise, it was known to cause performance issue:
      // - https://github.com/rapidsai/cudf/pull/6943
      // - https://github.com/rapidsai/cudf/pull/8277
      cuco::detail::open_addressing_ns::insert_if_n<primitive_probing_scheme::cg_size, cuco::detail::default_block_size()>
        <<<grid_size, cuco::detail::default_block_size(), 0, stream.value()>>>(
          thrust::counting_iterator<size_type>{0}, _build.num_rows(), thrust::counting_iterator<size_type>{0}, row_is_valid{row_bitmask_ptr}, set_ref);
    }
    else {
      cuco::detail::open_addressing_ns::insert_if_n<primitive_probing_scheme::cg_size, cuco::detail::default_block_size()>
        <<<grid_size, cuco::detail::default_block_size(), 0, stream.value()>>>(
          thrust::counting_iterator<size_type>{0}, _build.num_rows(), thrust::constant_iterator<bool>{true}, cuda::std::identity{}, set_ref);
    }    
  } 
  else {
    auto const build_has_nested_columns = cudf::has_nested_columns(_build);

    auto const d_build_hasher = row_hasher{_preprocessed_build}.device_hasher(nullate::DYNAMIC(build_has_nulls));
    auto const d_build_comparator = cudf::experimental::row::equality::self_comparator{_preprocessed_build};

    if(build_has_nested_columns) {
      auto d_build_nan_comparator = d_build_comparator.equal_to<true>(
        nullate::DYNAMIC{build_has_nulls}, compare_nulls, cudf::experimental::row::equality::nan_equal_physical_equality_comparator{});
      cuco::static_set_ref<key, cuda::thread_scope_device, row_comparator, nested_probing_scheme, storage_type_ref> set_ref{
        empty_sentinel_key, 
        d_build_nan_comparator, 
        nested_probing_scheme{d_build_hasher}, 
        cuda::thread_scope_device{}, 
        _bucket_storage.ref()};
      // Build hash table by inserting all rows from build table
      auto const build_iter = cudf::detail::make_counting_transform_iterator(
        size_type{0}, cuda::proclaim_return_type<rhs_index_type>([] __device__(auto idx) {
          return rhs_index_type{idx};
        }));

      auto const grid_size = cuco::detail::grid_size(_build.num_rows(), primitive_probing_scheme::cg_size);

      if (build_has_nulls && compare_nulls == null_equality::UNEQUAL) {
        auto const bitmask_buffer_and_ptr = build_row_bitmask(build, stream);
        auto const row_bitmask_ptr        = bitmask_buffer_and_ptr.second;

        // If the haystack table has nulls but they are compared unequal, don't insert them.
        // Otherwise, it was known to cause performance issue:
        // - https://github.com/rapidsai/cudf/pull/6943
        // - https://github.com/rapidsai/cudf/pull/8277
        cuco::detail::open_addressing_ns::insert_if_n<primitive_probing_scheme::cg_size, cuco::detail::default_block_size()>
          <<<grid_size, cuco::detail::default_block_size(), 0, stream.value()>>>(
            build_iter, _build.num_rows(), thrust::counting_iterator<size_type>{0}, row_is_valid{row_bitmask_ptr}, set_ref);
      }
      else {
        cuco::detail::open_addressing_ns::insert_if_n<primitive_probing_scheme::cg_size, cuco::detail::default_block_size()>
          <<<grid_size, cuco::detail::default_block_size(), 0, stream.value()>>>(
            build_iter, _build.num_rows(), thrust::constant_iterator<bool>{true}, cuda::std::identity{}, set_ref);
      }    
    }

    auto d_build_nan_comparator = d_build_comparator.equal_to<false>(
      nullate::DYNAMIC{build_has_nulls}, compare_nulls, cudf::experimental::row::equality::nan_equal_physical_equality_comparator{});
    cuco::static_set_ref<key, cuda::thread_scope_device, row_comparator, simple_probing_scheme, storage_type_ref> set_ref{
      empty_sentinel_key, 
      d_build_nan_comparator, 
      nested_probing_scheme{d_build_hasher}, 
      cuda::thread_scope_device{}, 
      _bucket_storage.ref()};
    // Build hash table by inserting all rows from build table
    auto const build_iter = cudf::detail::make_counting_transform_iterator(
      size_type{0}, cuda::proclaim_return_type<rhs_index_type>([] __device__(auto idx) {
        return rhs_index_type{idx};
      }));

    auto const grid_size = cuco::detail::grid_size(_build.num_rows(), primitive_probing_scheme::cg_size);

    if (build_has_nulls && compare_nulls == null_equality::UNEQUAL) {
      auto const bitmask_buffer_and_ptr = build_row_bitmask(build, stream);
      auto const row_bitmask_ptr        = bitmask_buffer_and_ptr.second;

      // If the haystack table has nulls but they are compared unequal, don't insert them.
      // Otherwise, it was known to cause performance issue:
      // - https://github.com/rapidsai/cudf/pull/6943
      // - https://github.com/rapidsai/cudf/pull/8277
      cuco::detail::open_addressing_ns::insert_if_n<primitive_probing_scheme::cg_size, cuco::detail::default_block_size()>
        <<<grid_size, cuco::detail::default_block_size(), 0, stream.value()>>>(
          build_iter, _build.num_rows(), thrust::counting_iterator<size_type>{0}, row_is_valid{row_bitmask_ptr}, set_ref);
    }
    else {
      cuco::detail::open_addressing_ns::insert_if_n<primitive_probing_scheme::cg_size, cuco::detail::default_block_size()>
        <<<grid_size, cuco::detail::default_block_size(), 0, stream.value()>>>(
          build_iter, _build.num_rows(), thrust::constant_iterator<bool>{true}, cuda::std::identity{}, set_ref);
    }    
  }
}

std::unique_ptr<rmm::device_uvector<cudf::size_type>> left_join::semi_join(cudf::table_view const& probe, rmm::cuda_stream_view stream) {
  auto const probe_has_nulls = has_nested_nulls(probe);
  auto empty_sentinel_key = cuco::empty_key{key{-1}};
  auto const preprocessed_probe =
    cudf::experimental::row::equality::preprocessed_table::create(probe, stream);

  auto const build_has_floating_point =
    std::any_of(_build.begin(), _build.end(), [](auto const& col) {
      return cudf::is_floating_point(col.type());
    });

  auto contained = rmm::device_uvector<bool>(probe.num_rows(), stream);

  if (cudf::is_primitive_row_op_compatible(_build) && !build_has_floating_point) {
    auto const d_probe_hasher =
      primitive_row_hasher{nullate::DYNAMIC{probe_has_nulls}, preprocessed_probe};
    auto const d_probe_build_comparator = cudf::row::primitive::row_equality_comparator{
      nullate::DYNAMIC{probe_has_nulls}, preprocessed_probe, _preprocessed_build, _nulls_equal};

    cuco::static_set_ref<key, cuda::thread_scope_device, primitive_row_comparator, primitive_probing_scheme, storage_type_ref> set_ref{
      empty_sentinel_key, 
      d_probe_build_comparator, 
      primitive_probing_scheme{d_probe_hasher}, 
      cuda::thread_scope_device{}, 
      _bucket_storage.ref()};

    auto const grid_size = cuco::detail::grid_size(_build.num_rows(), primitive_probing_scheme::cg_size);

    if (probe_has_nulls && _nulls_equal == null_equality::UNEQUAL) {
      auto const bitmask_buffer_and_ptr = build_row_bitmask(build, stream);
      auto const row_bitmask_ptr        = bitmask_buffer_and_ptr.second;

      cuco::detail::open_addressing_ns::contains_if_n<primitive_probing_scheme::cg_size, cuco::detail::default_block_size()>
        <<<grid_size, cuco::detail::default_block_size(), 0, stream.value()>>>(
          thrust::counting_iterator<size_type>{0}, probe.num_rows(), thrust::counting_iterator<size_type>{0}, row_is_valid{row_bitmask_ptr}, contained.begin(), set_ref);
    }
    else {
      cuco::detail::open_addressing_ns::contains_if_n<primitive_probing_scheme::cg_size, cuco::detail::default_block_size()>
        <<<grid_size, cuco::detail::default_block_size(), 0, stream.value()>>>(
          thrust::counting_iterator<size_type>{0}, _build.num_rows(), thrust::constant_iterator<bool>{true}, cuda::std::identity{}, contained.begin(), set_ref);
    }    
  }
}

}  // namespace detail
}  // namespace cudf
