#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2022, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "join_common_utils.cuh"
#include "join_common_utils.hpp"
#include "mixed_join_kernels.cuh"

#include <cudf/ast/detail/expression_parser.hpp>
#include <cudf/ast/expressions.hpp>
#include <cudf/detail/utilities/cuda.cuh>
#include <cudf/join.hpp>
#include <cudf/table/table.hpp>
#include <cudf/table/table_device_view.cuh>
#include <cudf/table/table_view.hpp>
#include <cudf/types.hpp>
#include <cudf/utilities/default_stream.hpp>
#include <cudf/utilities/span.hpp>

#include <rmm/cuda_stream_view.hpp>
#include <rmm/exec_policy.hpp>

#include <thrust/fill.h>
#include <thrust/scan.h>

#include <optional>
#include <utility>

namespace cudf {
namespace detail {

std::pair<std::unique_ptr<rmm::device_uvector<size_type>>,
          std::unique_ptr<rmm::device_uvector<size_type>>>
mixed_join(
  table_view const& left_equality,
  table_view const& right_equality,
  table_view const& left_conditional,
  table_view const& right_conditional,
  ast::expression const& binary_predicate,
  null_equality compare_nulls,
  join_kind join_type,
  std::optional<std::pair<std::size_t, device_span<size_type const>>> const& output_size_data,
  rmm::cuda_stream_view stream,
  rmm::mr::device_memory_resource* mr)
{
  CUDF_EXPECTS(left_conditional.num_rows() == left_equality.num_rows(),
               "The left conditional and equality tables must have the same number of rows.");
  CUDF_EXPECTS(right_conditional.num_rows() == right_equality.num_rows(),
               "The right conditional and equality tables must have the same number of rows.");

  CUDF_EXPECTS((join_type != join_kind::LEFT_SEMI_JOIN) && (join_type != join_kind::LEFT_ANTI_JOIN),
               "Left semi and anti joins should use mixed_join_semi.");

  auto const right_num_rows{right_conditional.num_rows()};
  auto const left_num_rows{left_conditional.num_rows()};
  auto const swap_tables = (join_type == join_kind::INNER_JOIN) && (right_num_rows > left_num_rows);

  // The "outer" table is the larger of the two tables. The kernels are
  // launched with one thread per row of the outer table, which also means that
  // it is the probe table for the hash
  auto const outer_num_rows{swap_tables ? right_num_rows : left_num_rows};

  // We can immediately filter out cases where the right table is empty. In
  // some cases, we return all the rows of the left table with a corresponding
  // null index for the right table; in others, we return an empty output.
  if (right_num_rows == 0) {
    switch (join_type) {
      // Left and full joins all return all the row indices from
      // left with a corresponding NULL from the right.
      case join_kind::LEFT_JOIN:
      case join_kind::FULL_JOIN: return get_trivial_left_join_indices(left_conditional, stream);
      // Inner joins return empty output because no matches can exist.
      case join_kind::INNER_JOIN:
        return std::pair(std::make_unique<rmm::device_uvector<size_type>>(0, stream, mr),
                         std::make_unique<rmm::device_uvector<size_type>>(0, stream, mr));
      default: CUDF_FAIL("Invalid join kind."); break;
    }
  } else if (left_num_rows == 0) {
    switch (join_type) {
      // Left and inner joins all return empty sets.
      case join_kind::LEFT_JOIN:
      case join_kind::INNER_JOIN:
        return std::pair(std::make_unique<rmm::device_uvector<size_type>>(0, stream, mr),
                         std::make_unique<rmm::device_uvector<size_type>>(0, stream, mr));
      // Full joins need to return the trivial complement.
      case join_kind::FULL_JOIN: {
        auto ret_flipped = get_trivial_left_join_indices(right_conditional, stream);
        return std::pair(std::move(ret_flipped.second), std::move(ret_flipped.first));
      }
      default: CUDF_FAIL("Invalid join kind."); break;
    }
  }

  // If evaluating the expression may produce null outputs we create a nullable
  // output column and follow the null-supporting expression evaluation code
  // path.
  auto const has_nulls =
    cudf::has_nulls(left_equality) || cudf::has_nulls(right_equality) ||
    binary_predicate.may_evaluate_null(left_conditional, right_conditional, stream);

  auto const parser = ast::detail::expression_parser{
    binary_predicate, left_conditional, right_conditional, has_nulls, stream, mr};
  CUDF_EXPECTS(parser.output_type().id() == type_id::BOOL8,
               "The expression must produce a boolean output.");

  // TODO: The non-conditional join impls start with a dictionary matching,
  // figure out what that is and what it's needed for (and if conditional joins
  // need to do the same).
  auto& probe     = swap_tables ? right_equality : left_equality;
  auto& build     = swap_tables ? left_equality : right_equality;
  auto probe_view = table_device_view::create(probe, stream);
  auto build_view = table_device_view::create(build, stream);
  row_equality equality_probe{
    cudf::nullate::DYNAMIC{has_nulls}, *probe_view, *build_view, compare_nulls};

  // Don't use multimap_type because we want a CG size of 1.
  mixed_multimap_type hash_table{
    compute_hash_table_size(build.num_rows()),
    cuco::sentinel::empty_key{std::numeric_limits<hash_value_type>::max()},
    cuco::sentinel::empty_value{cudf::detail::JoinNoneValue},
    stream.value(),
    detail::hash_table_allocator_type{default_allocator<char>{}, stream}};

  // TODO: To add support for nested columns we will need to flatten in many
  // places. However, this probably isn't worth adding any time soon since we
  // won't be able to support AST conditions for those types anyway.
  auto const row_bitmask = cudf::detail::bitmask_and(build, stream).first;
  build_join_hash_table(
    build, hash_table, compare_nulls, static_cast<bitmask_type const*>(row_bitmask.data()), stream);
  auto hash_table_view = hash_table.get_device_view();

  auto left_conditional_view  = table_device_view::create(left_conditional, stream);
  auto right_conditional_view = table_device_view::create(right_conditional, stream);

  // For inner joins we support optimizing the join by launching one thread for
  // whichever table is larger rather than always using the left table.
  detail::grid_1d const config(outer_num_rows, DEFAULT_JOIN_BLOCK_SIZE);
  auto const shmem_size_per_block = parser.shmem_per_thread * config.num_threads_per_block;
  join_kind const kernel_join_type =
    join_type == join_kind::FULL_JOIN ? join_kind::LEFT_JOIN : join_type;

  // If the join size data was not provided as an input, compute it here.
  std::size_t join_size;
  // Using an optional because we only need to allocate a new vector if one was
  // not passed as input, and rmm::device_uvector is not default constructible
  std::optional<rmm::device_uvector<size_type>> matches_per_row{};
  device_span<size_type const> matches_per_row_span{};

  if (output_size_data.has_value()) {
    join_size            = output_size_data->first;
    matches_per_row_span = output_size_data->second;
  } else {
    // Allocate storage for the counter used to get the size of the join output
    rmm::device_scalar<std::size_t> size(0, stream, mr);

    matches_per_row =
      rmm::device_uvector<size_type>{static_cast<std::size_t>(outer_num_rows), stream, mr};
    // Note that the view goes out of scope after this else statement, but the
    // data owned by matches_per_row stays alive so the data pointer is valid.
    auto mutable_matches_per_row_span = cudf::device_span<size_type>{
      matches_per_row->begin(), static_cast<std::size_t>(outer_num_rows)};
    matches_per_row_span = cudf::device_span<size_type const>{
      matches_per_row->begin(), static_cast<std::size_t>(outer_num_rows)};
    if (has_nulls) {
      compute_mixed_join_output_size<DEFAULT_JOIN_BLOCK_SIZE, true>
        <<<config.num_blocks, config.num_threads_per_block, shmem_size_per_block, stream.value()>>>(
          *left_conditional_view,
          *right_conditional_view,
          *probe_view,
          *build_view,
          equality_probe,
          kernel_join_type,
          hash_table_view,
          parser.device_expression_data,
          swap_tables,
          size.data(),
          mutable_matches_per_row_span);
    } else {
      compute_mixed_join_output_size<DEFAULT_JOIN_BLOCK_SIZE, false>
        <<<config.num_blocks, config.num_threads_per_block, shmem_size_per_block, stream.value()>>>(
          *left_conditional_view,
          *right_conditional_view,
          *probe_view,
          *build_view,
          equality_probe,
          kernel_join_type,
          hash_table_view,
          parser.device_expression_data,
          swap_tables,
          size.data(),
          mutable_matches_per_row_span);
    }
    join_size = size.value(stream);
  }

  // The initial early exit clauses guarantee that we will not reach this point
  // unless both the left and right tables are non-empty. Under that
  // constraint, neither left nor full joins can return an empty result since
  // at minimum we are guaranteed null matches for all non-matching rows. In
  // all other cases (inner, left semi, and left anti joins) if we reach this
  // point we can safely return an empty result.
  if (join_size == 0) {
    return std::pair(std::make_unique<rmm::device_uvector<size_type>>(0, stream, mr),
                     std::make_unique<rmm::device_uvector<size_type>>(0, stream, mr));
  }

  // Given the number of matches per row, we need to compute the offsets for insertion.
  auto join_result_offsets =
    rmm::device_uvector<size_type>{static_cast<std::size_t>(outer_num_rows), stream, mr};
  thrust::exclusive_scan(rmm::exec_policy{stream},
                         matches_per_row_span.begin(),
                         matches_per_row_span.end(),
                         join_result_offsets.begin());

  auto left_indices  = std::make_unique<rmm::device_uvector<size_type>>(join_size, stream, mr);
  auto right_indices = std::make_unique<rmm::device_uvector<size_type>>(join_size, stream, mr);

  auto const& join_output_l = left_indices->data();
  auto const& join_output_r = right_indices->data();

  if (has_nulls) {
    mixed_join<DEFAULT_JOIN_BLOCK_SIZE, true>
      <<<config.num_blocks, config.num_threads_per_block, shmem_size_per_block, stream.value()>>>(
        *left_conditional_view,
        *right_conditional_view,
        *probe_view,
        *build_view,
        equality_probe,
        kernel_join_type,
        hash_table_view,
        join_output_l,
        join_output_r,
        parser.device_expression_data,
        join_result_offsets.data(),
        swap_tables);
  } else {
    mixed_join<DEFAULT_JOIN_BLOCK_SIZE, false>
      <<<config.num_blocks, config.num_threads_per_block, shmem_size_per_block, stream.value()>>>(
        *left_conditional_view,
        *right_conditional_view,
        *probe_view,
        *build_view,
        equality_probe,
        kernel_join_type,
        hash_table_view,
        join_output_l,
        join_output_r,
        parser.device_expression_data,
        join_result_offsets.data(),
        swap_tables);
  }

  auto join_indices = std::pair(std::move(left_indices), std::move(right_indices));

  // For full joins, get the indices in the right table that were not joined to
  // by any row in the left table.
  if (join_type == join_kind::FULL_JOIN) {
    auto complement_indices = detail::get_left_join_indices_complement(
      join_indices.second, left_num_rows, right_num_rows, stream, mr);
    join_indices = detail::concatenate_vector_pairs(join_indices, complement_indices, stream);
  }
  return join_indices;
}

std::pair<std::size_t, std::unique_ptr<rmm::device_uvector<size_type>>>
compute_mixed_join_output_size(table_view const& left_equality,
                               table_view const& right_equality,
                               table_view const& left_conditional,
                               table_view const& right_conditional,
                               ast::expression const& binary_predicate,
                               null_equality compare_nulls,
                               join_kind join_type,
                               rmm::cuda_stream_view stream,
                               rmm::mr::device_memory_resource* mr)
{
  // Until we add logic to handle the number of non-matches in the right table,
  // full joins are not supported in this function. Note that this does not
  // prevent actually performing full joins since we do that by calculating the
  // left join and then concatenating the complementary right indices.
  CUDF_EXPECTS(join_type != join_kind::FULL_JOIN,
               "Size estimation is not available for full joins.");

  CUDF_EXPECTS(
    (join_type != join_kind::LEFT_SEMI_JOIN) && (join_type != join_kind::LEFT_ANTI_JOIN),
    "Left semi and anti join size estimation should use compute_mixed_join_output_size_semi.");

  CUDF_EXPECTS(left_conditional.num_rows() == left_equality.num_rows(),
               "The left conditional and equality tables must have the same number of rows.");
  CUDF_EXPECTS(right_conditional.num_rows() == right_equality.num_rows(),
               "The right conditional and equality tables must have the same number of rows.");

  auto const right_num_rows{right_conditional.num_rows()};
  auto const left_num_rows{left_conditional.num_rows()};
  auto const swap_tables = (join_type == join_kind::INNER_JOIN) && (right_num_rows > left_num_rows);

  // The "outer" table is the larger of the two tables. The kernels are
  // launched with one thread per row of the outer table, which also means that
  // it is the probe table for the hash
  auto const outer_num_rows{swap_tables ? right_num_rows : left_num_rows};

  auto matches_per_row = std::make_unique<rmm::device_uvector<size_type>>(
    static_cast<std::size_t>(outer_num_rows), stream, mr);
  auto matches_per_row_span = cudf::device_span<size_type>{
    matches_per_row->begin(), static_cast<std::size_t>(outer_num_rows)};

  // We can immediately filter out cases where one table is empty. In
  // some cases, we return all the rows of the other table with a corresponding
  // null index for the empty table; in others, we return an empty output.
  if (right_num_rows == 0) {
    switch (join_type) {
      // Left, left anti, and full all return all the row indices from left
      // with a corresponding NULL from the right.
      case join_kind::LEFT_JOIN:
      case join_kind::FULL_JOIN: {
        thrust::fill(matches_per_row->begin(), matches_per_row->end(), 1);
        return {left_num_rows, std::move(matches_per_row)};
      }
      // Inner and left semi joins return empty output because no matches can exist.
      case join_kind::INNER_JOIN: {
        thrust::fill(matches_per_row->begin(), matches_per_row->end(), 0);
        return {0, std::move(matches_per_row)};
      }
      default: CUDF_FAIL("Invalid join kind."); break;
    }
  } else if (left_num_rows == 0) {
    switch (join_type) {
      // Left, left anti, left semi, and inner joins all return empty sets.
      case join_kind::LEFT_JOIN:
      case join_kind::INNER_JOIN: {
        thrust::fill(matches_per_row->begin(), matches_per_row->end(), 0);
        return {0, std::move(matches_per_row)};
      }
      // Full joins need to return the trivial complement.
      case join_kind::FULL_JOIN: {
        thrust::fill(matches_per_row->begin(), matches_per_row->end(), 1);
        return {right_num_rows, std::move(matches_per_row)};
      }
      default: CUDF_FAIL("Invalid join kind."); break;
    }
  }

  // If evaluating the expression may produce null outputs we create a nullable
  // output column and follow the null-supporting expression evaluation code
  // path.
  auto const has_nulls =
    cudf::has_nulls(left_equality) || cudf::has_nulls(right_equality) ||
    binary_predicate.may_evaluate_null(left_conditional, right_conditional, stream);

  auto const parser = ast::detail::expression_parser{
    binary_predicate, left_conditional, right_conditional, has_nulls, stream, mr};
  CUDF_EXPECTS(parser.output_type().id() == type_id::BOOL8,
               "The expression must produce a boolean output.");

  // TODO: The non-conditional join impls start with a dictionary matching,
  // figure out what that is and what it's needed for (and if conditional joins
  // need to do the same).
  auto& probe     = swap_tables ? right_equality : left_equality;
  auto& build     = swap_tables ? left_equality : right_equality;
  auto probe_view = table_device_view::create(probe, stream);
  auto build_view = table_device_view::create(build, stream);
  row_equality equality_probe{
    cudf::nullate::DYNAMIC{has_nulls}, *probe_view, *build_view, compare_nulls};

  // Don't use multimap_type because we want a CG size of 1.
  mixed_multimap_type hash_table{
    compute_hash_table_size(build.num_rows()),
    cuco::sentinel::empty_key{std::numeric_limits<hash_value_type>::max()},
    cuco::sentinel::empty_value{cudf::detail::JoinNoneValue},
    stream.value(),
    detail::hash_table_allocator_type{default_allocator<char>{}, stream}};

  // TODO: To add support for nested columns we will need to flatten in many
  // places. However, this probably isn't worth adding any time soon since we
  // won't be able to support AST conditions for those types anyway.
  auto const row_bitmask = cudf::detail::bitmask_and(build, stream).first;
  build_join_hash_table(
    build, hash_table, compare_nulls, static_cast<bitmask_type const*>(row_bitmask.data()), stream);
  auto hash_table_view = hash_table.get_device_view();

  auto left_conditional_view  = table_device_view::create(left_conditional, stream);
  auto right_conditional_view = table_device_view::create(right_conditional, stream);

  // For inner joins we support optimizing the join by launching one thread for
  // whichever table is larger rather than always using the left table.
  detail::grid_1d const config(outer_num_rows, DEFAULT_JOIN_BLOCK_SIZE);
  auto const shmem_size_per_block = parser.shmem_per_thread * config.num_threads_per_block;

  // Allocate storage for the counter used to get the size of the join output
  rmm::device_scalar<std::size_t> size(0, stream, mr);

  // Determine number of output rows without actually building the output to simply
  // find what the size of the output will be.
  if (has_nulls) {
    compute_mixed_join_output_size<DEFAULT_JOIN_BLOCK_SIZE, true>
      <<<config.num_blocks, config.num_threads_per_block, shmem_size_per_block, stream.value()>>>(
        *left_conditional_view,
        *right_conditional_view,
        *probe_view,
        *build_view,
        equality_probe,
        join_type,
        hash_table_view,
        parser.device_expression_data,
        swap_tables,
        size.data(),
        matches_per_row_span);
  } else {
    compute_mixed_join_output_size<DEFAULT_JOIN_BLOCK_SIZE, false>
      <<<config.num_blocks, config.num_threads_per_block, shmem_size_per_block, stream.value()>>>(
        *left_conditional_view,
        *right_conditional_view,
        *probe_view,
        *build_view,
        equality_probe,
        join_type,
        hash_table_view,
        parser.device_expression_data,
        swap_tables,
        size.data(),
        matches_per_row_span);
  }

  return {size.value(stream), std::move(matches_per_row)};
}

}  // namespace detail

std::pair<std::unique_ptr<rmm::device_uvector<size_type>>,
          std::unique_ptr<rmm::device_uvector<size_type>>>
mixed_inner_join(
  table_view const& left_equality,
  table_view const& right_equality,
  table_view const& left_conditional,
  table_view const& right_conditional,
  ast::expression const& binary_predicate,
  null_equality compare_nulls,
  std::optional<std::pair<std::size_t, device_span<size_type const>>> const output_size_data,
  rmm::mr::device_memory_resource* mr)
{
  CUDF_FUNC_RANGE();
  return detail::mixed_join(left_equality,
                            right_equality,
                            left_conditional,
                            right_conditional,
                            binary_predicate,
                            compare_nulls,
                            detail::join_kind::INNER_JOIN,
                            output_size_data,
                            cudf::get_default_stream(),
                            mr);
}

std::pair<std::size_t, std::unique_ptr<rmm::device_uvector<size_type>>> mixed_inner_join_size(
  table_view const& left_equality,
  table_view const& right_equality,
  table_view const& left_conditional,
  table_view const& right_conditional,
  ast::expression const& binary_predicate,
  null_equality compare_nulls,
  rmm::mr::device_memory_resource* mr)
{
  CUDF_FUNC_RANGE();
  return detail::compute_mixed_join_output_size(left_equality,
                                                right_equality,
                                                left_conditional,
                                                right_conditional,
                                                binary_predicate,
                                                compare_nulls,
                                                detail::join_kind::INNER_JOIN,
                                                cudf::get_default_stream(),
                                                mr);
}

std::pair<std::unique_ptr<rmm::device_uvector<size_type>>,
          std::unique_ptr<rmm::device_uvector<size_type>>>
mixed_left_join(
  table_view const& left_equality,
  table_view const& right_equality,
  table_view const& left_conditional,
  table_view const& right_conditional,
  ast::expression const& binary_predicate,
  null_equality compare_nulls,
  std::optional<std::pair<std::size_t, device_span<size_type const>>> const output_size_data,
  rmm::mr::device_memory_resource* mr)
{
  CUDF_FUNC_RANGE();
  return detail::mixed_join(left_equality,
                            right_equality,
                            left_conditional,
                            right_conditional,
                            binary_predicate,
                            compare_nulls,
                            detail::join_kind::LEFT_JOIN,
                            output_size_data,
                            cudf::get_default_stream(),
                            mr);
}

std::pair<std::size_t, std::unique_ptr<rmm::device_uvector<size_type>>> mixed_left_join_size(
  table_view const& left_equality,
  table_view const& right_equality,
  table_view const& left_conditional,
  table_view const& right_conditional,
  ast::expression const& binary_predicate,
  null_equality compare_nulls,
  rmm::mr::device_memory_resource* mr)
{
  CUDF_FUNC_RANGE();
  return detail::compute_mixed_join_output_size(left_equality,
                                                right_equality,
                                                left_conditional,
                                                right_conditional,
                                                binary_predicate,
                                                compare_nulls,
                                                detail::join_kind::LEFT_JOIN,
                                                cudf::get_default_stream(),
                                                mr);
}

std::pair<std::unique_ptr<rmm::device_uvector<size_type>>,
          std::unique_ptr<rmm::device_uvector<size_type>>>
mixed_full_join(
  table_view const& left_equality,
  table_view const& right_equality,
  table_view const& left_conditional,
  table_view const& right_conditional,
  ast::expression const& binary_predicate,
  null_equality compare_nulls,
  std::optional<std::pair<std::size_t, device_span<size_type const>>> const output_size_data,
  rmm::mr::device_memory_resource* mr)
{
  CUDF_FUNC_RANGE();
  return detail::mixed_join(left_equality,
                            right_equality,
                            left_conditional,
                            right_conditional,
                            binary_predicate,
                            compare_nulls,
                            detail::join_kind::FULL_JOIN,
                            output_size_data,
                            cudf::get_default_stream(),
                            mr);
}

}  // namespace cudf
