/*
 * Copyright (c) 2020-2022, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <join/join_common_utils.hpp>

#include <cudf/detail/gather.hpp>
#include <cudf/detail/iterator.cuh>
#include <cudf/detail/nvtx/ranges.hpp>
#include <cudf/detail/search.hpp>
#include <cudf/dictionary/detail/update_keys.hpp>
#include <cudf/join.hpp>
#include <cudf/table/table.hpp>
#include <cudf/utilities/default_stream.hpp>
#include <cudf/utilities/error.hpp>

#include <rmm/cuda_stream_view.hpp>
#include <rmm/device_uvector.hpp>
#include <rmm/exec_policy.hpp>

#include <thrust/copy.h>
#include <thrust/distance.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/sequence.h>
#include <thrust/transform.h>

namespace cudf {
namespace detail {

std::unique_ptr<rmm::device_uvector<cudf::size_type>> left_semi_anti_join(
  join_kind const kind,
  cudf::table_view const& left_keys,
  cudf::table_view const& right_keys,
  null_equality compare_nulls,
  rmm::cuda_stream_view stream,
  rmm::mr::device_memory_resource* mr = rmm::mr::get_current_device_resource())
{
  CUDF_EXPECTS(0 != left_keys.num_columns(), "Left table is empty");
  CUDF_EXPECTS(0 != right_keys.num_columns(), "Right table is empty");

  if (is_trivial_join(left_keys, right_keys, kind)) {
    return std::make_unique<rmm::device_uvector<cudf::size_type>>(0, stream, mr);
  }
  if ((join_kind::LEFT_ANTI_JOIN == kind) && (0 == right_keys.num_rows())) {
    auto result =
      std::make_unique<rmm::device_uvector<cudf::size_type>>(left_keys.num_rows(), stream, mr);
    thrust::sequence(rmm::exec_policy(stream), result->begin(), result->end());
    return result;
  }

  // Materialize a `flagged` boolean array to generate a gather map.
  // Previously, the gather map was generated directly without this array but by calling to
  // `map.contains` inside the `thrust::copy_if` kernel. However, that led to increasing register
  // usage and reducing performance, as reported here: https://github.com/rapidsai/cudf/pull/10511.
  auto const flagged =
    cudf::detail::contains(right_keys, left_keys, compare_nulls, nan_equality::ALL_EQUAL, stream);

  auto const left_num_rows = left_keys.num_rows();
  auto gather_map =
    std::make_unique<rmm::device_uvector<cudf::size_type>>(left_num_rows, stream, mr);

  // gather_map_end will be the end of valid data in gather_map
  auto gather_map_end =
    thrust::copy_if(rmm::exec_policy(stream),
                    thrust::counting_iterator<size_type>(0),
                    thrust::counting_iterator<size_type>(left_num_rows),
                    gather_map->begin(),
                    [kind, d_flagged = flagged.begin()] __device__(size_type const idx) {
                      return *(d_flagged + idx) == (kind == join_kind::LEFT_SEMI_JOIN);
                    });

  gather_map->resize(thrust::distance(gather_map->begin(), gather_map_end), stream);
  return gather_map;
}

}  // namespace detail

std::unique_ptr<rmm::device_uvector<cudf::size_type>> left_semi_join(
  cudf::table_view const& left,
  cudf::table_view const& right,
  null_equality compare_nulls,
  rmm::mr::device_memory_resource* mr)
{
  CUDF_FUNC_RANGE();
  return detail::left_semi_anti_join(
    detail::join_kind::LEFT_SEMI_JOIN, left, right, compare_nulls, cudf::get_default_stream(), mr);
}

std::unique_ptr<rmm::device_uvector<cudf::size_type>> left_anti_join(
  cudf::table_view const& left,
  cudf::table_view const& right,
  null_equality compare_nulls,
  rmm::mr::device_memory_resource* mr)
{
  CUDF_FUNC_RANGE();
  return detail::left_semi_anti_join(
    detail::join_kind::LEFT_ANTI_JOIN, left, right, compare_nulls, cudf::get_default_stream(), mr);
}

}  // namespace cudf
