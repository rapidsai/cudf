#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2020-2021, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <hash/concurrent_unordered_map.cuh>
#include <join/join_common_utils.hpp>

#include <cudf/column/column_factories.hpp>
#include <cudf/detail/gather.hpp>
#include <cudf/detail/null_mask.hpp>
#include <cudf/detail/nvtx/ranges.hpp>
#include <cudf/detail/structs/utilities.hpp>
#include <cudf/dictionary/detail/update_keys.hpp>
#include <cudf/join.hpp>
#include <cudf/table/table.hpp>
#include <cudf/utilities/error.hpp>

#include <rmm/cuda_stream_view.hpp>
#include <rmm/device_uvector.hpp>
#include <rmm/exec_policy.hpp>

#include <thrust/copy.h>
#include <thrust/distance.h>
#include <thrust/sequence.h>

namespace cudf {
namespace detail {

std::unique_ptr<rmm::device_uvector<cudf::size_type>> left_semi_anti_join(
  join_kind const kind,
  cudf::table_view const& left_keys,
  cudf::table_view const& right_keys,
  null_equality compare_nulls,
  rmm::cuda_stream_view stream,
  rmm::mr::device_memory_resource* mr = rmm::mr::get_current_device_resource())
{
  CUDF_EXPECTS(0 != left_keys.num_columns(), "Left table is empty");
  CUDF_EXPECTS(0 != right_keys.num_columns(), "Right table is empty");

  if (is_trivial_join(left_keys, right_keys, kind)) {
    return std::make_unique<rmm::device_uvector<cudf::size_type>>(0, stream, mr);
  }
  if ((join_kind::LEFT_ANTI_JOIN == kind) && (0 == right_keys.num_rows())) {
    auto result =
      std::make_unique<rmm::device_uvector<cudf::size_type>>(left_keys.num_rows(), stream, mr);
    thrust::sequence(rmm::exec_policy(stream), result->begin(), result->end());
    return result;
  }

  auto const left_num_rows  = left_keys.num_rows();
  auto const right_num_rows = right_keys.num_rows();

  // flatten structs for the right and left and use that for the hash table
  auto right_flattened_tables = structs::detail::flatten_nested_columns(
    right_keys, {}, {}, structs::detail::column_nullability::FORCE);
  auto left_flattened_tables = structs::detail::flatten_nested_columns(
    left_keys, {}, {}, structs::detail::column_nullability::FORCE);

  auto right_flattened_keys = right_flattened_tables.flattened_columns();
  auto left_flattened_keys  = left_flattened_tables.flattened_columns();

  // Only care about existence, so we'll use an unordered map (other joins need a multimap)
  using hash_table_type = concurrent_unordered_map<cudf::size_type, bool, row_hash, row_equality>;

  // Create hash table containing all keys found in right table
  auto right_rows_d            = table_device_view::create(right_flattened_keys, stream);
  size_t const hash_table_size = compute_hash_table_size(right_num_rows);
  row_hash hash_build{cudf::nullate::YES{}, *right_rows_d};
  row_equality equality_build{cudf::nullate::YES{}, *right_rows_d, *right_rows_d, compare_nulls};

  // Going to join it with left table
  auto left_rows_d = table_device_view::create(left_flattened_keys, stream);
  row_hash hash_probe{cudf::nullate::YES{}, *left_rows_d};
  row_equality equality_probe{cudf::nullate::YES{}, *left_rows_d, *right_rows_d, compare_nulls};

  auto hash_table_ptr = hash_table_type::create(hash_table_size,
                                                stream,
                                                std::numeric_limits<bool>::max(),
                                                std::numeric_limits<cudf::size_type>::max(),
                                                hash_build,
                                                equality_build);
  auto hash_table     = *hash_table_ptr;

  // if compare_nulls == UNEQUAL, we can simply ignore any rows that
  // contain a NULL in any column as they will never compare to equal.
  auto const row_bitmask = (compare_nulls == null_equality::EQUAL)
                             ? rmm::device_buffer{}
                             : cudf::detail::bitmask_and(right_flattened_keys, stream).first;
  // skip rows that are null here.
  thrust::for_each_n(
    rmm::exec_policy(stream),
    thrust::make_counting_iterator<size_type>(0),
    right_num_rows,
    [hash_table, row_bitmask = static_cast<bitmask_type const*>(row_bitmask.data())] __device__(
      size_type idx) mutable {
      if (!row_bitmask || cudf::bit_is_set(row_bitmask, idx)) {
        hash_table.insert(thrust::make_pair(idx, true));
      }
    });

  //
  // Now we have a hash table, we need to iterate over the rows of the left table
  // and check to see if they are contained in the hash table
  //

  // For semi join we want contains to be true, for anti join we want contains to be false
  bool const join_type_boolean = (kind == join_kind::LEFT_SEMI_JOIN);

  auto gather_map =
    std::make_unique<rmm::device_uvector<cudf::size_type>>(left_num_rows, stream, mr);

  // gather_map_end will be the end of valid data in gather_map
  auto gather_map_end = thrust::copy_if(
    rmm::exec_policy(stream),
    thrust::make_counting_iterator<size_type>(0),
    thrust::make_counting_iterator<size_type>(left_num_rows),
    gather_map->begin(),
    [hash_table, join_type_boolean, hash_probe, equality_probe] __device__(size_type idx) {
      auto pos = hash_table.find(idx, hash_probe, equality_probe);
      return (pos != hash_table.end()) == join_type_boolean;
    });

  auto join_size = thrust::distance(gather_map->begin(), gather_map_end);
  gather_map->resize(join_size, stream);
  return gather_map;
}

/**
 * @brief  Performs a left semi or anti join on the specified columns of two
 * tables (left, right)
 *
 * The semi and anti joins only return data from the left table. A left semi join
 * returns rows that exist in the right table, a left anti join returns rows
 * that do not exist in the right table.
 *
 * The basic approach is to create a hash table containing the contents of the right
 * table and then select only rows that exist (or don't exist) to be included in
 * the return set.
 *
 * @throws cudf::logic_error if number of columns in either `left` or `right` table is 0
 * @throws cudf::logic_error if number of returned columns is 0
 * @throws cudf::logic_error if number of elements in `right_on` and `left_on` are not equal
 *
 * @param kind          Indicates whether to do LEFT_SEMI_JOIN or LEFT_ANTI_JOIN
 * @param left          The left table
 * @param right         The right table
 * @param left_on       The column indices from `left` to join on.
 *                      The column from `left` indicated by `left_on[i]`
 *                      will be compared against the column from `right`
 *                      indicated by `right_on[i]`.
 * @param right_on      The column indices from `right` to join on.
 *                      The column from `right` indicated by `right_on[i]`
 *                      will be compared against the column from `left`
 *                      indicated by `left_on[i]`.
 * @param compare_nulls Controls whether null join-key values should match or not.
 * @param stream        CUDA stream used for device memory operations and kernel launches.
 * @param mr            Device memory resource to used to allocate the returned table
 *
 * @returns             Result of joining `left` and `right` tables on the columns
 *                      specified by `left_on` and `right_on`.
 */
std::unique_ptr<cudf::table> left_semi_anti_join(
  join_kind const kind,
  cudf::table_view const& left,
  cudf::table_view const& right,
  std::vector<cudf::size_type> const& left_on,
  std::vector<cudf::size_type> const& right_on,
  null_equality compare_nulls,
  rmm::cuda_stream_view stream,
  rmm::mr::device_memory_resource* mr = rmm::mr::get_current_device_resource())
{
  CUDF_EXPECTS(left_on.size() == right_on.size(), "Mismatch in number of columns to be joined on");

  if ((left_on.empty() || right_on.empty()) || is_trivial_join(left, right, kind)) {
    return empty_like(left);
  }

  if ((join_kind::LEFT_ANTI_JOIN == kind) && (0 == right.num_rows())) {
    // Everything matches, just copy the proper columns from the left table
    return std::make_unique<table>(left, stream, mr);
  }

  // Make sure any dictionary columns have matched key sets.
  // This will return any new dictionary columns created as well as updated table_views.
  auto matched = cudf::dictionary::detail::match_dictionaries(
    {left.select(left_on), right.select(right_on)},
    stream,
    rmm::mr::get_current_device_resource());  // temporary objects returned

  auto const left_selected  = matched.second.front();
  auto const right_selected = matched.second.back();

  auto gather_vector =
    left_semi_anti_join(kind, left_selected, right_selected, compare_nulls, stream);

  // wrapping the device vector with a column view allows calling the non-iterator
  // version of detail::gather, improving compile time by 10% and reducing the
  // object file size by 2.2x without affecting performance
  auto gather_map = column_view(data_type{type_id::INT32},
                                static_cast<size_type>(gather_vector->size()),
                                gather_vector->data(),
                                nullptr,
                                0);

  auto const left_updated = scatter_columns(left_selected, left_on, left);
  return cudf::detail::gather(left_updated,
                              gather_map,
                              out_of_bounds_policy::DONT_CHECK,
                              negative_index_policy::NOT_ALLOWED,
                              stream,
                              mr);
}

}  // namespace detail

std::unique_ptr<cudf::table> left_semi_join(cudf::table_view const& left,
                                            cudf::table_view const& right,
                                            std::vector<cudf::size_type> const& left_on,
                                            std::vector<cudf::size_type> const& right_on,
                                            null_equality compare_nulls,
                                            rmm::mr::device_memory_resource* mr)
{
  CUDF_FUNC_RANGE();
  return detail::left_semi_anti_join(detail::join_kind::LEFT_SEMI_JOIN,
                                     left,
                                     right,
                                     left_on,
                                     right_on,
                                     compare_nulls,
                                     rmm::cuda_stream_default,
                                     mr);
}

std::unique_ptr<rmm::device_uvector<cudf::size_type>> left_semi_join(
  cudf::table_view const& left,
  cudf::table_view const& right,
  null_equality compare_nulls,
  rmm::mr::device_memory_resource* mr)
{
  CUDF_FUNC_RANGE();
  return detail::left_semi_anti_join(
    detail::join_kind::LEFT_SEMI_JOIN, left, right, compare_nulls, rmm::cuda_stream_default, mr);
}

std::unique_ptr<cudf::table> left_anti_join(cudf::table_view const& left,
                                            cudf::table_view const& right,
                                            std::vector<cudf::size_type> const& left_on,
                                            std::vector<cudf::size_type> const& right_on,
                                            null_equality compare_nulls,
                                            rmm::mr::device_memory_resource* mr)
{
  CUDF_FUNC_RANGE();
  return detail::left_semi_anti_join(detail::join_kind::LEFT_ANTI_JOIN,
                                     left,
                                     right,
                                     left_on,
                                     right_on,
                                     compare_nulls,
                                     rmm::cuda_stream_default,
                                     mr);
}

std::unique_ptr<rmm::device_uvector<cudf::size_type>> left_anti_join(
  cudf::table_view const& left,
  cudf::table_view const& right,
  null_equality compare_nulls,
  rmm::mr::device_memory_resource* mr)
{
  CUDF_FUNC_RANGE();
  return detail::left_semi_anti_join(
    detail::join_kind::LEFT_ANTI_JOIN, left, right, compare_nulls, rmm::cuda_stream_default, mr);
}

}  // namespace cudf
