/*
 * Copyright (c) 2019, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <cudf/cudf.h>
#include <utilities/cudf_utils.h>
#include <cudf/utilities/error.hpp>

#include <table/legacy/device_table.cuh>
#include <table/legacy/device_table_row_operators.cuh>

#include <rmm/thrust_rmm_allocator.h>

namespace cudf {

bool is_sorted(cudf::table const& table,
                       std::vector<int8_t> const& descending,
                       bool nulls_are_smallest = false)                       
{
  hipStream_t stream = 0;
  bool sorted = false;
  auto ord_vect_size = descending.size();

  if (ord_vect_size != 0)
  {
      CUDF_EXPECTS(static_cast <unsigned int>(table.num_columns()) == descending.size(), "Number of columns in the table doesn't match the vector descending's size .\n");
  }
  
  if (table.num_columns() == 0 || table.num_rows() == 0)
  {
      return true;
  }

  auto exec = rmm::exec_policy(stream)->on(stream);
  auto device_input_table = device_table::create(table);
  bool const nullable = cudf::has_nulls(table);

  cudf::size_type nrows = table.num_rows();
 
  if (nullable)
  { 
      auto ineq_op = row_inequality_comparator<true>(*device_input_table, nulls_are_smallest, 
                                                        (ord_vect_size != 0)? (rmm::device_vector<int8_t> (descending)).data().get() : nullptr);
      sorted = thrust::is_sorted (exec, thrust::make_counting_iterator(0), thrust::make_counting_iterator(nrows), ineq_op);
  }
  else
  {
      auto ineq_op = row_inequality_comparator<false>(*device_input_table, nulls_are_smallest, 
                                                        (ord_vect_size != 0)? (rmm::device_vector<int8_t> (descending)).data().get() : nullptr);
      sorted = thrust::is_sorted (exec, thrust::make_counting_iterator(0), thrust::make_counting_iterator(nrows), ineq_op);
  }

  return sorted;
}
}
