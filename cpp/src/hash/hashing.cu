/*
 * Copyright (c) 2019, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */
#include <cudf/detail/hashing.hpp>
#include <cudf/copying.hpp>
#include <cudf/column/column_factories.hpp>
#include <cudf/table/table_device_view.cuh>
#include <cudf/utilities/nvtx_utils.hpp>
#include <cudf/detail/utilities/hash_functions.cuh>
#include <cudf/detail/utilities/cuda.cuh>
#include <cudf/table/row_operators.cuh>
#include <cudf/detail/scatter.cuh>
#include <cudf/detail/gather.cuh>
#include <cudf/detail/nvtx/ranges.hpp>
#include <cudf/partitioning.hpp>

#include <thrust/tabulate.h>

namespace cudf {
namespace detail {

std::unique_ptr<column> hash(table_view const& input,
                             std::vector<uint32_t> const& initial_hash,
                             rmm::mr::device_memory_resource* mr,
                             hipStream_t stream)
{
  // TODO this should be UINT32
  auto output = make_numeric_column(data_type(INT32), input.num_rows());

  // Return early if there's nothing to hash
  if (input.num_columns() == 0 || input.num_rows() == 0) {
    return output;
  }

  bool const nullable = has_nulls(input);
  auto const device_input = table_device_view::create(input, stream);
  auto output_view = output->mutable_view();

  // Compute the hash value for each row depending on the specified hash function
  if (!initial_hash.empty()) {
    CUDF_EXPECTS(initial_hash.size() == size_t(input.num_columns()),
      "Expected same size of initial hash values as number of columns");
    auto device_initial_hash = rmm::device_vector<uint32_t>(initial_hash);

    if (nullable) {
      thrust::tabulate(rmm::exec_policy(stream)->on(stream),
          output_view.begin<int32_t>(), output_view.end<int32_t>(),
          experimental::row_hasher_initial_values<MurmurHash3_32, true>(
              *device_input, device_initial_hash.data().get()));
    } else {
      thrust::tabulate(rmm::exec_policy(stream)->on(stream),
          output_view.begin<int32_t>(), output_view.end<int32_t>(),
          experimental::row_hasher_initial_values<MurmurHash3_32, false>(
              *device_input, device_initial_hash.data().get()));
    }
  } else {
    if (nullable) {
      thrust::tabulate(rmm::exec_policy(stream)->on(stream),
          output_view.begin<int32_t>(), output_view.end<int32_t>(),
          experimental::row_hasher<MurmurHash3_32, true>(*device_input));
    } else {
      thrust::tabulate(rmm::exec_policy(stream)->on(stream),
          output_view.begin<int32_t>(), output_view.end<int32_t>(),
          experimental::row_hasher<MurmurHash3_32, false>(*device_input));
    }
  }

  return output;
}

}  // namespace detail


std::unique_ptr<column> hash(table_view const& input,
                             std::vector<uint32_t> const& initial_hash,
                             rmm::mr::device_memory_resource* mr)
{
  CUDF_FUNC_RANGE();
  return detail::hash(input, initial_hash, mr);
}

}  // namespace cudf
