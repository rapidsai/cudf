#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */
#include <cudf/detail/hashing.hpp>
#include <cudf/copying.hpp>
#include <cudf/column/column_factories.hpp>
#include <cudf/table/table_device_view.cuh>
#include <cudf/utilities/nvtx_utils.hpp>
#include <cudf/detail/utilities/hash_functions.cuh>
#include <cudf/detail/utilities/cuda.cuh>
#include <cudf/table/row_operators.cuh>
#include <cudf/detail/scatter.hpp>

#include <thrust/tabulate.h>

namespace cudf {

namespace {

/** 
 * @brief  Functor to map a hash value to a particular 'bin' or partition number
 * that uses the modulo operation.
 */
template <typename hash_value_t>
class modulo_partitioner
{
 public:
  modulo_partitioner(size_type num_partitions) : divisor{num_partitions} {}

  __device__
  size_type operator()(hash_value_t hash_value) const {
    return hash_value % divisor;
  }

 private:
  const size_type divisor;
};

template <typename T>
bool is_power_two(T number) {
  return (0 == (number & (number - 1)));
}

/** 
 * @brief  Functor to map a hash value to a particular 'bin' or partition number
 * that uses a bitwise mask. Only works when num_partitions is a power of 2.
 *
 * For n % d, if d is a power of two, then it can be computed more efficiently via 
 * a single bitwise AND as:
 * n & (d - 1)
 */
template <typename hash_value_t>
class bitwise_partitioner
{
 public:
  bitwise_partitioner(size_type num_partitions) : mask{(num_partitions - 1)} {
    assert(is_power_two(num_partitions));
  }

  __device__
  size_type operator()(hash_value_t hash_value) const {
    return hash_value & mask; // hash_value & (num_partitions - 1)
  }

 private:
  const size_type mask;
};

/* --------------------------------------------------------------------------*/
/** 
 * @brief Computes which partition each row of a device_table will belong to based
   on hashing each row, and applying a partition function to the hash value. 
   Records the size of each partition for each thread block as well as the global
   size of each partition across all thread blocks.
 * 
 * @param[in] the_table The table whose rows will be partitioned
 * @param[in] num_rows The number of rows in the table
 * @param[in] num_partitions The number of partitions to divide the rows into
 * @param[in] the_partitioner The functor that maps a rows hash value to a partition number
 * @param[out] row_partition_numbers Array that holds which partition each row belongs to
 * @param[out] block_partition_sizes Array that holds the size of each partition for each block,
 * i.e., { {block0 partition0 size, block1 partition0 size, ...}, 
         {block0 partition1 size, block1 partition1 size, ...},
         ...
         {block0 partition(num_partitions-1) size, block1 partition(num_partitions -1) size, ...} }
 * @param[out] global_partition_sizes The number of rows in each partition.
 */
/* ----------------------------------------------------------------------------*/
template <class row_hasher_t, typename partitioner_type>
__global__
void compute_row_partition_numbers(row_hasher_t the_hasher,
                                   const size_type num_rows,
                                   const size_type num_partitions,
                                   const partitioner_type the_partitioner,
                                   size_type * __restrict__ row_partition_numbers,
                                   size_type * __restrict__ block_partition_sizes,
                                   size_type * __restrict__ global_partition_sizes)
{
  // Accumulate histogram of the size of each partition in shared memory
  extern __shared__ size_type shared_partition_sizes[];

  size_type row_number = threadIdx.x + blockIdx.x * blockDim.x;

  // Initialize local histogram
  size_type partition_number = threadIdx.x;
  while(partition_number < num_partitions)
  {
    shared_partition_sizes[partition_number] = 0;
    partition_number += blockDim.x;
  }

  __syncthreads();

  // Compute the hash value for each row, store it to the array of hash values
  // and compute the partition to which the hash value belongs and increment
  // the shared memory counter for that partition
  while( row_number < num_rows)
  {
    const hash_value_type row_hash_value = the_hasher(row_number);

    const size_type partition_number = the_partitioner(row_hash_value);

    row_partition_numbers[row_number] = partition_number;

    atomicAdd(&(shared_partition_sizes[partition_number]), size_type(1));

    row_number += blockDim.x * gridDim.x;
  }

  __syncthreads();

  // Flush shared memory histogram to global memory
  partition_number = threadIdx.x;
  while(partition_number < num_partitions)
  {
    const size_type block_partition_size = shared_partition_sizes[partition_number];

    // Update global size of each partition
    atomicAdd(&global_partition_sizes[partition_number], block_partition_size);

    // Record the size of this partition in this block
    const size_type write_location = partition_number * gridDim.x + blockIdx.x;
    block_partition_sizes[write_location] = block_partition_size;
    partition_number += blockDim.x;
  }
}

/* --------------------------------------------------------------------------*/
/** 
 * @brief  Given an array of partition numbers, computes the final output location
   for each element in the output such that all rows with the same partition are 
   contiguous in memory.
 * 
 * @param row_partition_numbers The array that records the partition number for each row
 * @param num_rows The number of rows
 * @param num_partitions THe number of partitions
 * @param[out] block_partition_offsets Array that holds the offset of each partition for each thread block,
 * i.e., { {block0 partition0 offset, block1 partition0 offset, ...}, 
         {block0 partition1 offset, block1 partition1 offset, ...},
         ...
         {block0 partition(num_partitions-1) offset, block1 partition(num_partitions -1) offset, ...} }
 */
/* ----------------------------------------------------------------------------*/
__global__ 
void compute_row_output_locations(size_type * __restrict__ row_partition_numbers, 
                                  const size_type num_rows,
                                  const size_type num_partitions,
                                  size_type * __restrict__ block_partition_offsets)
{
  // Shared array that holds the offset of this blocks partitions in 
  // global memory
  extern __shared__ size_type shared_partition_offsets[];

  // Initialize array of this blocks offsets from global array
  size_type partition_number= threadIdx.x;
  while(partition_number < num_partitions)
  {
    shared_partition_offsets[partition_number] = block_partition_offsets[partition_number * gridDim.x + blockIdx.x];
    partition_number += blockDim.x;
  }
  __syncthreads();

  size_type row_number = threadIdx.x + blockIdx.x * blockDim.x;

  // Get each row's partition number, and get it's output location by 
  // incrementing block's offset counter for that partition number
  // and store the row's output location in-place
  while( row_number < num_rows )
  {
    // Get partition number of this row
    const size_type partition_number = row_partition_numbers[row_number];

    // Get output location based on partition number by incrementing the corresponding
    // partition offset for this block
    const size_type row_output_location = atomicAdd(&(shared_partition_offsets[partition_number]), size_type(1));

    // Store the row's output location in-place
    row_partition_numbers[row_number] = row_output_location;

    row_number += blockDim.x * gridDim.x;
  }
}

template <bool has_nulls>
std::pair<std::unique_ptr<experimental::table>, std::vector<size_type>>
hash_partition_table(table_view const& input,
                     table_view const &table_to_hash,
                     const size_type num_partitions,
                     rmm::mr::device_memory_resource* mr,
                     hipStream_t stream)
{
  auto const num_rows = table_to_hash.num_rows();

  constexpr size_type BLOCK_SIZE = 256;
  constexpr size_type ROWS_PER_THREAD = 1;
  constexpr size_type rows_per_block = BLOCK_SIZE * ROWS_PER_THREAD;
  auto grid_size = util::div_rounding_up_safe(num_rows, rows_per_block);

  // Allocate array to hold which partition each row belongs to
  auto row_partition_numbers = rmm::device_vector<size_type>(num_rows);

  // Array to hold the size of each partition computed by each block
  //  i.e., { {block0 partition0 size, block1 partition0 size, ...}, 
  //          {block0 partition1 size, block1 partition1 size, ...},
  //          ...
  //          {block0 partition(num_partitions-1) size, block1 partition(num_partitions -1) size, ...} }
  auto block_partition_sizes = rmm::device_vector<size_type>(grid_size * num_partitions);

  // Holds the total number of rows in each partition
  auto global_partition_sizes = rmm::device_vector<size_type>(num_partitions, size_type{0});

  auto const device_input = table_device_view::create(table_to_hash, stream);
  auto const hasher = experimental::row_hasher<MurmurHash3_32, has_nulls>(*device_input);

  // If the number of partitions is a power of two, we can compute the partition 
  // number of each row more efficiently with bitwise operations
  if (is_power_two(num_partitions)) {
    // Determines how the mapping between hash value and partition number is computed
    using partitioner_type = bitwise_partitioner<hash_value_type>;

    // Computes which partition each row belongs to by hashing the row and performing
    // a partitioning operator on the hash value. Also computes the number of
    // rows in each partition both for each thread block as well as across all blocks
    compute_row_partition_numbers
        <<<grid_size, BLOCK_SIZE, num_partitions * sizeof(size_type), stream>>>(
            hasher, num_rows, num_partitions,
            partitioner_type(num_partitions),
            row_partition_numbers.data().get(),
            block_partition_sizes.data().get(),
            global_partition_sizes.data().get());
  } else {
    // Determines how the mapping between hash value and partition number is computed
    using partitioner_type = modulo_partitioner<hash_value_type>;

    // Computes which partition each row belongs to by hashing the row and performing
    // a partitioning operator on the hash value. Also computes the number of
    // rows in each partition both for each thread block as well as across all blocks
    compute_row_partition_numbers
        <<<grid_size, BLOCK_SIZE, num_partitions * sizeof(size_type), stream>>>(
            hasher, num_rows, num_partitions,
            partitioner_type(num_partitions),
            row_partition_numbers.data().get(),
            block_partition_sizes.data().get(),
            global_partition_sizes.data().get());
  }

  // Compute exclusive scan of all blocks' partition sizes in-place to determine 
  // the starting point for each blocks portion of each partition in the output
  cudf::size_type * scanned_block_partition_sizes{block_partition_sizes.data().get()};
  thrust::exclusive_scan(rmm::exec_policy(stream)->on(stream),
                         block_partition_sizes.begin(), 
                         block_partition_sizes.end(), 
                         scanned_block_partition_sizes);

  // Compute exclusive scan of size of each partition to determine offset location
  // of each partition in final output.
  // TODO This can be done independently on a separate stream
  size_type * scanned_global_partition_sizes{global_partition_sizes.data().get()};
  thrust::exclusive_scan(rmm::exec_policy(stream)->on(stream),
                         global_partition_sizes.begin(), 
                         global_partition_sizes.end(),
                         scanned_global_partition_sizes);

  // Copy the result of the exlusive scan to the output offsets array
  // to indicate the starting point for each partition in the output
  std::vector<size_type> partition_offsets(num_partitions);
  CUDA_TRY(hipMemcpyAsync(partition_offsets.data(), 
                           scanned_global_partition_sizes, 
                           num_partitions * sizeof(size_type),
                           hipMemcpyDeviceToHost,
                           stream));

  // Compute the output location for each row in-place based on it's 
  // partition number such that each partition will be contiguous in memory
  size_type * row_output_locations{row_partition_numbers.data().get()};
  compute_row_output_locations
      <<<grid_size, BLOCK_SIZE, num_partitions * sizeof(size_type), stream>>>
          (row_output_locations, num_rows, num_partitions, scanned_block_partition_sizes);

  auto scatter_map = column_view{data_type{INT32}, num_rows, row_output_locations};
  auto output = experimental::detail::scatter(input, scatter_map, input, false, mr, stream);

  return std::make_pair(std::move(output), std::move(partition_offsets));
}

// Add a wrapper around nvtx to automatically pop the range when the function scope ends
struct nvtx_raii {
  nvtx_raii(char const* name, nvtx::color color) { nvtx::range_push(name, color); }
  ~nvtx_raii() { nvtx::range_pop(); }
};

}  // namespace

namespace detail {

std::pair<std::unique_ptr<experimental::table>, std::vector<size_type>>
hash_partition(table_view const& input,
               std::vector<size_type> const& columns_to_hash,
               int num_partitions,
               rmm::mr::device_memory_resource* mr,
               hipStream_t stream)
{
  // Push/pop nvtx range around the scope of this function
  nvtx_raii("CUDF_HASH_PARTITION", nvtx::PARTITION_COLOR);

  auto table_to_hash = input.select(columns_to_hash);

  // Return empty result if there are no partitions or nothing to hash
  if (num_partitions <= 0 || input.num_rows() == 0 || table_to_hash.num_columns() == 0) {
    return std::make_pair(experimental::empty_like(input), std::vector<size_type>{});
  }

  if (has_nulls(table_to_hash)) {
    return hash_partition_table<true>(
        input, table_to_hash, num_partitions, mr, stream);
  } else {
    return hash_partition_table<false>(
        input, table_to_hash, num_partitions, mr, stream);
  }
}

std::unique_ptr<column> hash(table_view const& input,
                             std::vector<uint32_t> const& initial_hash,
                             rmm::mr::device_memory_resource* mr,
                             hipStream_t stream)
{
  // TODO this should be UINT32
  auto output = make_numeric_column(data_type(INT32), input.num_rows());

  // Return early if there's nothing to hash
  if (input.num_columns() == 0 || input.num_rows() == 0) {
    return output;
  }

  bool const nullable = has_nulls(input);
  auto const device_input = table_device_view::create(input, stream);
  auto output_view = output->mutable_view();

  // Compute the hash value for each row depending on the specified hash function
  if (!initial_hash.empty()) {
    CUDF_EXPECTS(initial_hash.size() == size_t(input.num_columns()),
      "Expected same size of initial hash values as number of columns");
    auto device_initial_hash = rmm::device_vector<uint32_t>(initial_hash);

    if (nullable) {
      thrust::tabulate(rmm::exec_policy(stream)->on(stream),
          output_view.begin<int32_t>(), output_view.end<int32_t>(),
          experimental::row_hasher_initial_values<MurmurHash3_32, true>(
              *device_input, device_initial_hash.data().get()));
    } else {
      thrust::tabulate(rmm::exec_policy(stream)->on(stream),
          output_view.begin<int32_t>(), output_view.end<int32_t>(),
          experimental::row_hasher_initial_values<MurmurHash3_32, false>(
              *device_input, device_initial_hash.data().get()));
    }
  } else {
    if (nullable) {
      thrust::tabulate(rmm::exec_policy(stream)->on(stream),
          output_view.begin<int32_t>(), output_view.end<int32_t>(),
          experimental::row_hasher<MurmurHash3_32, true>(*device_input));
    } else {
      thrust::tabulate(rmm::exec_policy(stream)->on(stream),
          output_view.begin<int32_t>(), output_view.end<int32_t>(),
          experimental::row_hasher<MurmurHash3_32, false>(*device_input));
    }
  }

  return output;
}

}  // namespace detail

std::pair<std::unique_ptr<experimental::table>, std::vector<size_type>>
hash_partition(table_view const& input,
               std::vector<size_type> const& columns_to_hash,
               int num_partitions,
               rmm::mr::device_memory_resource* mr)
{
  return detail::hash_partition(input, columns_to_hash, num_partitions, mr);
}

std::unique_ptr<column> hash(table_view const& input,
                             std::vector<uint32_t> const& initial_hash,
                             rmm::mr::device_memory_resource* mr)
{
  return detail::hash(input, initial_hash, mr);
}

}  // namespace cudf
