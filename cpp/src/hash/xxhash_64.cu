#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2023-2024, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */
#include <cudf/column/column_factories.hpp>
#include <cudf/detail/nvtx/ranges.hpp>
#include <cudf/detail/utilities/algorithm.cuh>
#include <cudf/hashing/detail/hashing.hpp>
#include <cudf/hashing/detail/xxhash_64.cuh>
#include <cudf/table/table_device_view.cuh>
#include <cudf/utilities/memory_resource.hpp>
#include <cudf/utilities/span.hpp>

#include <rmm/cuda_stream_view.hpp>
#include <rmm/exec_policy.hpp>

#include <cuda/std/limits>
#include <thrust/tabulate.h>

namespace cudf {
namespace hashing {
namespace detail {

namespace {

using hash_value_type = uint64_t;

/**
 * @brief Computes the hash value of a row in the given table.
 *
 * @tparam Nullate A cudf::nullate type describing whether to check for nulls.
 */
template <typename Nullate>
class device_row_hasher {
 public:
  device_row_hasher(Nullate nulls, table_device_view const& t, hash_value_type seed)
    : _check_nulls(nulls), _table(t), _seed(seed)
  {
  }

  __device__ auto operator()(size_type row_index) const noexcept
  {
    return cudf::detail::accumulate(
      _table.begin(),
      _table.end(),
      _seed,
      [row_index, nulls = _check_nulls] __device__(auto hash, auto column) {
        return cudf::type_dispatcher(
          column.type(), element_hasher_adapter{}, column, row_index, nulls, hash);
      });
  }

  /**
   * @brief Computes the hash value of an element in the given column.
   */
  class element_hasher_adapter {
   public:
    template <typename T, CUDF_ENABLE_IF(column_device_view::has_element_accessor<T>())>
    __device__ hash_value_type operator()(column_device_view const& col,
                                          size_type const row_index,
                                          Nullate const _check_nulls,
                                          hash_value_type const _seed) const noexcept
    {
      if (_check_nulls && col.is_null(row_index)) {
        return cuda::std::numeric_limits<hash_value_type>::max();
      }
      auto const hasher = XXHash_64<T>{_seed};
      return hasher(col.element<T>(row_index));
    }

    template <typename T, CUDF_ENABLE_IF(not column_device_view::has_element_accessor<T>())>
    __device__ hash_value_type operator()(column_device_view const&,
                                          size_type const,
                                          Nullate const,
                                          hash_value_type const) const noexcept
    {
      CUDF_UNREACHABLE("Unsupported type for XXHash_64");
    }
  };

  Nullate const _check_nulls;
  table_device_view const _table;
  hash_value_type const _seed;
};

}  // namespace

std::unique_ptr<column> xxhash_64(table_view const& input,
                                  uint64_t seed,
                                  rmm::cuda_stream_view stream,
                                  rmm::device_async_resource_ref mr)
{
  auto output = make_numeric_column(data_type(type_to_id<hash_value_type>()),
                                    input.num_rows(),
                                    mask_state::UNALLOCATED,
                                    stream,
                                    mr);

  // Return early if there's nothing to hash
  if (input.num_columns() == 0 || input.num_rows() == 0) { return output; }

  bool const nullable   = has_nulls(input);
  auto const input_view = table_device_view::create(input, stream);
  auto output_view      = output->mutable_view();

  // Compute the hash value for each row
  thrust::tabulate(rmm::exec_policy(stream),
                   output_view.begin<hash_value_type>(),
                   output_view.end<hash_value_type>(),
                   device_row_hasher(nullable, *input_view, seed));

  return output;
}

}  // namespace detail

std::unique_ptr<column> xxhash_64(table_view const& input,
                                  uint64_t seed,
                                  rmm::cuda_stream_view stream,
                                  rmm::device_async_resource_ref mr)
{
  CUDF_FUNC_RANGE();
  return detail::xxhash_64(input, seed, stream, mr);
}

}  // namespace hashing
}  // namespace cudf
