#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2023-2024, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */
#include <cudf/column/column_factories.hpp>
#include <cudf/detail/nvtx/ranges.hpp>
#include <cudf/detail/utilities/algorithm.cuh>
#include <cudf/hashing/detail/hash_functions.cuh>
#include <cudf/hashing/detail/hashing.hpp>
#include <cudf/table/table_device_view.cuh>
#include <cudf/utilities/memory_resource.hpp>
#include <cudf/utilities/span.hpp>

#include <rmm/cuda_stream_view.hpp>
#include <rmm/exec_policy.hpp>

#include <cuco/hash_functions.cuh>
#include <thrust/tabulate.h>

namespace cudf {
namespace hashing {
namespace detail {

namespace {

using hash_value_type = uint64_t;

template <typename Key>
struct XXHash_64 : public cuco::xxhash_64<Key> {
  __device__ hash_value_type operator()(Key const& key) const
  {
    return cuco::xxhash_64<Key>::operator()(key);
  }

  template <typename Extent>
  __device__ hash_value_type compute_hash(cuda::std::byte const* bytes, Extent size) const
  {
    return cuco::xxhash_64<Key>::compute_hash(bytes, size);
  }
};

template <>
hash_value_type __device__ inline XXHash_64<bool>::operator()(bool const& key) const
{
  return this->compute_hash(reinterpret_cast<cuda::std::byte const*>(&key), sizeof(key));
}

template <>
hash_value_type __device__ inline XXHash_64<float>::operator()(float const& key) const
{
  return cuco::xxhash_64<float>::operator()(normalize_nans(key));
}

template <>
hash_value_type __device__ inline XXHash_64<double>::operator()(double const& key) const
{
  return cuco::xxhash_64<double>::operator()(normalize_nans(key));
}

template <>
hash_value_type __device__ inline XXHash_64<cudf::string_view>::operator()(
  cudf::string_view const& key) const
{
  return this->compute_hash(reinterpret_cast<cuda::std::byte const*>(key.data()), key.size_bytes());
}

template <>
hash_value_type __device__ inline XXHash_64<numeric::decimal32>::operator()(
  numeric::decimal32 const& key) const
{
  auto const val = key.value();
  auto const len = sizeof(val);
  return this->compute_hash(reinterpret_cast<cuda::std::byte const*>(&val), len);
}

template <>
hash_value_type __device__ inline XXHash_64<numeric::decimal64>::operator()(
  numeric::decimal64 const& key) const
{
  auto const val = key.value();
  auto const len = sizeof(val);
  return this->compute_hash(reinterpret_cast<cuda::std::byte const*>(&val), len);
}

template <>
hash_value_type __device__ inline XXHash_64<numeric::decimal128>::operator()(
  numeric::decimal128 const& key) const
{
  auto const val = key.value();
  auto const len = sizeof(val);
  return this->compute_hash(reinterpret_cast<cuda::std::byte const*>(&val), len);
}

/**
 * @brief Computes the hash value of a row in the given table.
 *
 * @tparam Nullate A cudf::nullate type describing whether to check for nulls.
 */
template <typename Nullate>
class device_row_hasher {
 public:
  device_row_hasher(Nullate nulls, table_device_view const& t, hash_value_type seed)
    : _check_nulls(nulls), _table(t), _seed(seed)
  {
  }

  __device__ auto operator()(size_type row_index) const noexcept
  {
    return cudf::detail::accumulate(
      _table.begin(),
      _table.end(),
      _seed,
      [row_index, nulls = _check_nulls] __device__(auto hash, auto column) {
        return cudf::type_dispatcher(
          column.type(), element_hasher_adapter{}, column, row_index, nulls, hash);
      });
  }

  /**
   * @brief Computes the hash value of an element in the given column.
   */
  class element_hasher_adapter {
   public:
    template <typename T, CUDF_ENABLE_IF(column_device_view::has_element_accessor<T>())>
    __device__ hash_value_type operator()(column_device_view const& col,
                                          size_type const row_index,
                                          Nullate const _check_nulls,
                                          hash_value_type const _seed) const noexcept
    {
      if (_check_nulls && col.is_null(row_index)) {
        return std::numeric_limits<hash_value_type>::max();
      }
      auto const hasher = XXHash_64<T>{_seed};
      return hasher(col.element<T>(row_index));
    }

    template <typename T, CUDF_ENABLE_IF(not column_device_view::has_element_accessor<T>())>
    __device__ hash_value_type operator()(column_device_view const&,
                                          size_type const,
                                          Nullate const,
                                          hash_value_type const) const noexcept
    {
      CUDF_UNREACHABLE("Unsupported type for XXHash_64");
    }
  };

  Nullate const _check_nulls;
  table_device_view const _table;
  hash_value_type const _seed;
};

}  // namespace

std::unique_ptr<column> xxhash_64(table_view const& input,
                                  uint64_t seed,
                                  rmm::cuda_stream_view stream,
                                  rmm::device_async_resource_ref mr)
{
  auto output = make_numeric_column(data_type(type_to_id<hash_value_type>()),
                                    input.num_rows(),
                                    mask_state::UNALLOCATED,
                                    stream,
                                    mr);

  // Return early if there's nothing to hash
  if (input.num_columns() == 0 || input.num_rows() == 0) { return output; }

  bool const nullable   = has_nulls(input);
  auto const input_view = table_device_view::create(input, stream);
  auto output_view      = output->mutable_view();

  // Compute the hash value for each row
  thrust::tabulate(rmm::exec_policy(stream),
                   output_view.begin<hash_value_type>(),
                   output_view.end<hash_value_type>(),
                   device_row_hasher(nullable, *input_view, seed));

  return output;
}

}  // namespace detail

std::unique_ptr<column> xxhash_64(table_view const& input,
                                  uint64_t seed,
                                  rmm::cuda_stream_view stream,
                                  rmm::device_async_resource_ref mr)
{
  CUDF_FUNC_RANGE();
  return detail::xxhash_64(input, seed, stream, mr);
}

}  // namespace hashing
}  // namespace cudf
