/*
 * Copyright (c) 2019-2022, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <strings/regex/dispatcher.hpp>
#include <strings/regex/regex.cuh>
#include <strings/utilities.hpp>

#include <cudf/column/column.hpp>
#include <cudf/column/column_device_view.cuh>
#include <cudf/detail/iterator.cuh>
#include <cudf/detail/nvtx/ranges.hpp>
#include <cudf/strings/detail/strings_column_factories.cuh>
#include <cudf/strings/extract.hpp>
#include <cudf/strings/string_view.cuh>
#include <cudf/strings/strings_column_view.hpp>
#include <cudf/utilities/span.hpp>

#include <rmm/cuda_stream_view.hpp>

#include <thrust/execution_policy.h>
#include <thrust/for_each.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/iterator/permutation_iterator.h>
#include <thrust/pair.h>

namespace cudf {
namespace strings {
namespace detail {

namespace {

using string_index_pair = thrust::pair<const char*, size_type>;

/**
 * @brief This functor handles extracting strings by applying the compiled regex pattern
 * and creating string_index_pairs for all the substrings.
 *
 * @tparam stack_size Correlates to the regex instructions state to maintain for each string.
 *         Each instruction requires a fixed amount of overhead data.
 */
template <int stack_size>
struct extract_fn {
  reprog_device prog;
  column_device_view const d_strings;
  cudf::detail::device_2dspan<string_index_pair> d_indices;

  __device__ void operator()(size_type idx)
  {
    auto const groups = prog.group_counts();
    auto d_output     = d_indices[idx];

    if (d_strings.is_valid(idx)) {
      auto const d_str = d_strings.element<string_view>(idx);
      int32_t begin    = 0;
      int32_t end      = -1;  // handles empty strings automatically
      if (prog.find<stack_size>(idx, d_str, begin, end) > 0) {
        for (auto col_idx = 0; col_idx < groups; ++col_idx) {
          auto const extracted = prog.extract<stack_size>(idx, d_str, begin, end, col_idx);
          d_output[col_idx]    = [&] {
            if (!extracted) return string_index_pair{nullptr, 0};
            auto const offset = d_str.byte_offset((*extracted).first);
            return string_index_pair{d_str.data() + offset,
                                     d_str.byte_offset((*extracted).second) - offset};
          }();
        }
        return;
      }
    }

    // if null row or no match found, fill the output with null entries
    thrust::fill(thrust::seq, d_output.begin(), d_output.end(), string_index_pair{nullptr, 0});
  }
};

struct extract_dispatch_fn {
  reprog_device d_prog;

  template <int stack_size>
  void operator()(column_device_view const& d_strings,
                  cudf::detail::device_2dspan<string_index_pair>& d_indices,
                  rmm::cuda_stream_view stream)
  {
    thrust::for_each_n(rmm::exec_policy(stream),
                       thrust::make_counting_iterator<size_type>(0),
                       d_strings.size(),
                       extract_fn<stack_size>{d_prog, d_strings, d_indices});
  }
};
}  // namespace

//
std::unique_ptr<table> extract(
  strings_column_view const& input,
  std::string const& pattern,
  regex_flags const flags,
  rmm::cuda_stream_view stream,
  rmm::mr::device_memory_resource* mr = rmm::mr::get_current_device_resource())
{
  // compile regex into device object
  auto d_prog =
    reprog_device::create(pattern, flags, get_character_flags_table(), input.size(), stream);

  auto const groups = d_prog->group_counts();
  CUDF_EXPECTS(groups > 0, "Group indicators not found in regex pattern");

  auto indices = rmm::device_uvector<string_index_pair>(input.size() * groups, stream);
  auto d_indices =
    cudf::detail::device_2dspan<string_index_pair>(indices.data(), input.size(), groups);

  auto const d_strings = column_device_view::create(input.parent(), stream);
  regex_dispatcher(*d_prog, extract_dispatch_fn{*d_prog}, *d_strings, d_indices, stream);

  // build a result column for each group
  std::vector<std::unique_ptr<column>> results(groups);
  auto make_strings_lambda = [&](size_type column_index) {
    // this iterator transposes the extract results into column order
    auto indices_itr =
      thrust::make_permutation_iterator(indices.begin(),
                                        cudf::detail::make_counting_transform_iterator(
                                          0, [column_index, groups] __device__(size_type idx) {
                                            return (idx * groups) + column_index;
                                          }));
    return make_strings_column(indices_itr, indices_itr + input.size(), stream, mr);
  };

  std::transform(thrust::make_counting_iterator<size_type>(0),
                 thrust::make_counting_iterator<size_type>(groups),
                 results.begin(),
                 make_strings_lambda);

  return std::make_unique<table>(std::move(results));
}

}  // namespace detail

// external API

std::unique_ptr<table> extract(strings_column_view const& strings,
                               std::string const& pattern,
                               regex_flags const flags,
                               rmm::mr::device_memory_resource* mr)
{
  CUDF_FUNC_RANGE();
  return detail::extract(strings, pattern, flags, rmm::cuda_stream_default, mr);
}

}  // namespace strings
}  // namespace cudf
