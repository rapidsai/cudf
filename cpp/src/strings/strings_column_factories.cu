#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019-2020, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <cudf/column/column.hpp>
#include <cudf/column/column_factories.hpp>
#include <cudf/detail/nvtx/ranges.hpp>
#include <cudf/detail/valid_if.cuh>
#include <cudf/strings/detail/utilities.hpp>
#include <cudf/utilities/error.hpp>
#include <strings/utilities.cuh>

#include <rmm/cuda_stream_view.hpp>
#include <rmm/device_vector.hpp>
#include <rmm/exec_policy.hpp>

#include <thrust/for_each.h>
#include <thrust/transform_reduce.h>

namespace cudf {

// Create a strings-type column from vector of pointer/size pairs
std::unique_ptr<column> make_strings_column(
  const rmm::device_vector<thrust::pair<const char*, size_type>>& strings,
  rmm::cuda_stream_view stream,
  rmm::mr::device_memory_resource* mr)
{
  CUDF_FUNC_RANGE();
  size_type strings_count = strings.size();
  if (strings_count == 0) return strings::detail::make_empty_strings_column(stream, mr);

  auto d_strings = strings.data().get();

  // check total size is not too large for cudf column
  auto size_checker = [d_strings] __device__(size_t idx) {
    auto item = d_strings[idx];
    return (item.first != nullptr) ? item.second : 0;
  };
  size_t bytes = thrust::transform_reduce(rmm::exec_policy(stream),
                                          thrust::make_counting_iterator<size_t>(0),
                                          thrust::make_counting_iterator<size_t>(strings_count),
                                          size_checker,
                                          0,
                                          thrust::plus<size_t>());
  CUDF_EXPECTS(bytes < std::numeric_limits<size_type>::max(),
               "total size of strings is too large for cudf column");

  // build offsets column from the strings sizes
  auto offsets_transformer = [d_strings] __device__(size_type idx) {
    thrust::pair<const char*, size_type> item = d_strings[idx];
    return (item.first != nullptr ? static_cast<int32_t>(item.second) : 0);
  };
  auto offsets_transformer_itr = thrust::make_transform_iterator(
    thrust::make_counting_iterator<size_type>(0), offsets_transformer);
  auto offsets_column = strings::detail::make_offsets_child_column(
    offsets_transformer_itr, offsets_transformer_itr + strings_count, stream, mr);
  auto offsets_view = offsets_column->view();
  auto d_offsets    = offsets_view.data<int32_t>();

  // create null mask
  auto new_nulls = detail::valid_if(
    thrust::make_counting_iterator<size_type>(0),
    thrust::make_counting_iterator<size_type>(strings_count),
    [d_strings] __device__(size_type idx) { return d_strings[idx].first != nullptr; },
    stream,
    mr);
  auto null_count = new_nulls.second;
  rmm::device_buffer null_mask{0, stream, mr};
  if (null_count > 0) null_mask = std::move(new_nulls.first);

  // build chars column
  auto chars_column =
    strings::detail::create_chars_child_column(strings_count, null_count, bytes, stream, mr);
  auto chars_view = chars_column->mutable_view();
  auto d_chars    = chars_view.data<char>();
  thrust::for_each_n(rmm::exec_policy(stream),
                     thrust::make_counting_iterator<size_type>(0),
                     strings_count,
                     [d_strings, d_offsets, d_chars] __device__(size_type idx) {
                       // place individual strings
                       auto item = d_strings[idx];
                       if (item.first != nullptr)
                         memcpy(d_chars + d_offsets[idx], item.first, item.second);
                     });

  return make_strings_column(strings_count,
                             std::move(offsets_column),
                             std::move(chars_column),
                             null_count,
                             std::move(null_mask),
                             stream,
                             mr);
}

struct string_view_to_pair {
  string_view null_placeholder;
  string_view_to_pair(string_view n) : null_placeholder(n) {}
  __device__ thrust::pair<const char*, size_type> operator()(const string_view& i)
  {
    return (i.data() == null_placeholder.data())
             ? thrust::pair<const char*, size_type>{nullptr, 0}
             : thrust::pair<const char*, size_type>{i.data(), i.size_bytes()};
  }
};

// Create a strings-type column from vector of string_view
std::unique_ptr<column> make_strings_column(const rmm::device_vector<string_view>& string_views,
                                            const string_view null_placeholder,
                                            rmm::cuda_stream_view stream,
                                            rmm::mr::device_memory_resource* mr)
{
  auto it_pair =
    thrust::make_transform_iterator(string_views.begin(), string_view_to_pair{null_placeholder});
  const rmm::device_vector<thrust::pair<const char*, size_type>> dev_strings(
    it_pair, it_pair + string_views.size());
  return make_strings_column(dev_strings, stream, mr);
}

// Create a strings-type column from device vector of chars and vector of offsets.
std::unique_ptr<column> make_strings_column(const rmm::device_vector<char>& strings,
                                            const rmm::device_vector<size_type>& offsets,
                                            const rmm::device_vector<bitmask_type>& valid_mask,
                                            size_type null_count,
                                            rmm::cuda_stream_view stream,
                                            rmm::mr::device_memory_resource* mr)
{
  CUDF_FUNC_RANGE();
  size_type num_strings = offsets.size() - 1;
  if (num_strings == 0) return strings::detail::make_empty_strings_column(stream, mr);

  CUDF_EXPECTS(null_count < num_strings, "null strings column not yet supported");
  if (null_count > 0) {
    CUDF_EXPECTS(!valid_mask.empty(), "Cannot have null elements without a null mask.");
  }

  size_type bytes = offsets.back();
  CUDF_EXPECTS(bytes >= 0, "invalid offsets vector");

  // build offsets column -- this is the number of strings + 1
  auto offsets_column = make_numeric_column(
    data_type{type_id::INT32}, num_strings + 1, mask_state::UNALLOCATED, stream, mr);
  auto offsets_view = offsets_column->mutable_view();
  CUDA_TRY(hipMemcpyAsync(offsets_view.data<int32_t>(),
                           offsets.data().get(),
                           (num_strings + 1) * sizeof(int32_t),
                           hipMemcpyDeviceToDevice,
                           stream.value()));
  // build null bitmask
  rmm::device_buffer null_mask{
    valid_mask.data().get(),
    valid_mask.size() *
      sizeof(bitmask_type)};  // Or this works too: sizeof(typename
                              // std::remove_reference_t<decltype(valid_mask)>::value_type)
  // Following give the incorrect value of 8 instead of 4 because of smart references:
  // sizeof(valid_mask[0]), sizeof(decltype(valid_mask.front()))

  // build chars column
  auto chars_column =
    strings::detail::create_chars_child_column(num_strings, null_count, bytes, stream, mr);
  auto chars_view = chars_column->mutable_view();
  CUDA_TRY(hipMemcpyAsync(chars_view.data<char>(),
                           strings.data().get(),
                           bytes,
                           hipMemcpyDeviceToDevice,
                           stream.value()));

  return make_strings_column(num_strings,
                             std::move(offsets_column),
                             std::move(chars_column),
                             null_count,
                             std::move(null_mask),
                             stream,
                             mr);
}

// Create strings column from host vectors
std::unique_ptr<column> make_strings_column(const std::vector<char>& strings,
                                            const std::vector<size_type>& offsets,
                                            const std::vector<bitmask_type>& null_mask,
                                            size_type null_count,
                                            rmm::cuda_stream_view stream,
                                            rmm::mr::device_memory_resource* mr)
{
  rmm::device_vector<char> d_strings{strings};
  rmm::device_vector<size_type> d_offsets{offsets};
  rmm::device_vector<bitmask_type> d_null_mask{null_mask};

  return make_strings_column(d_strings, d_offsets, d_null_mask, null_count, stream, mr);
}

//
std::unique_ptr<column> make_strings_column(size_type num_strings,
                                            std::unique_ptr<column> offsets_column,
                                            std::unique_ptr<column> chars_column,
                                            size_type null_count,
                                            rmm::device_buffer&& null_mask,
                                            rmm::cuda_stream_view stream,
                                            rmm::mr::device_memory_resource* mr)
{
  if (null_count > 0) CUDF_EXPECTS(null_mask.size() > 0, "Column with nulls must be nullable.");
  CUDF_EXPECTS(num_strings == offsets_column->size() - 1,
               "Invalid offsets column size for strings column.");
  CUDF_EXPECTS(offsets_column->null_count() == 0, "Offsets column should not contain nulls");
  CUDF_EXPECTS(chars_column->null_count() == 0, "Chars column should not contain nulls");

  std::vector<std::unique_ptr<column>> children;
  children.emplace_back(std::move(offsets_column));
  children.emplace_back(std::move(chars_column));
  return std::make_unique<column>(data_type{type_id::STRING},
                                  num_strings,
                                  rmm::device_buffer{0, stream, mr},
                                  null_mask,
                                  null_count,
                                  std::move(children));
}

}  // namespace cudf
