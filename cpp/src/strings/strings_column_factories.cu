#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <bitmask/legacy/valid_if.cuh>
#include <cudf/column/column_factories.hpp>
#include <cudf/column/column.hpp>
#include <cudf/legacy/functions.h>
#include <cudf/null_mask.hpp>
#include <cudf/utilities/error.hpp>
#include "./utilities.hpp"
#include "./utilities.cuh"

#include <rmm/thrust_rmm_allocator.h>
#include <thrust/transform_reduce.h>
#include <thrust/transform_scan.h>
#include <thrust/for_each.h>


namespace cudf {

// Create a strings-type column from vector of pointer/size pairs
std::unique_ptr<column> make_strings_column(
    const rmm::device_vector<thrust::pair<const char*,size_type>>& strings,
    hipStream_t stream,
    rmm::mr::device_memory_resource* mr)
{
    size_type strings_count = strings.size();
    // maybe a separate factory for creating null strings-column
    CUDF_EXPECTS(strings_count > 0, "must specify at least one pair");

    auto execpol = rmm::exec_policy(stream);
    auto d_strings = strings.data().get();

    // check total size is not too large for cudf column
    size_t bytes = thrust::transform_reduce( execpol->on(stream),
        thrust::make_counting_iterator<size_t>(0),
        thrust::make_counting_iterator<size_t>(strings_count),
        [d_strings] __device__ (size_t idx) {
            auto item = d_strings[idx];
            return (item.first!=nullptr) ? item.second : 0;
        },
        0, thrust::plus<size_t>());
    CUDF_EXPECTS( bytes < std::numeric_limits<size_type>::max(), "total size of strings is too large for cudf column" );

    // build offsets column from the strings sizes
    auto offsets_transformer = [d_strings] __device__ (size_type idx) {
            thrust::pair<const char*,size_type> item = d_strings[idx];
            return ( item.first!=nullptr ? static_cast<int32_t>(item.second) : 0 );
        };
    auto offsets_transformer_itr = thrust::make_transform_iterator( thrust::make_counting_iterator<size_type>(0), offsets_transformer );
    auto offsets_column = strings::detail::make_offsets_child_column(offsets_transformer_itr,
                                               offsets_transformer_itr+strings_count,
                                               mr, stream);
    auto offsets_view = offsets_column->view();
    auto d_offsets = offsets_view.data<int32_t>();

    // create null mask
    auto valid_mask = strings::detail::make_null_mask(strings_count,
        [d_strings] __device__ (size_type idx) { return d_strings[idx].first!=nullptr; },
        mr, stream);
    auto null_count = valid_mask.second;
    rmm::device_buffer null_mask = valid_mask.first;

    // build chars column
    auto chars_column = strings::detail::create_chars_child_column( strings_count, null_count, bytes, mr, stream );
    auto chars_view = chars_column->mutable_view();
    auto d_chars = chars_view.data<char>();
    thrust::for_each_n(execpol->on(stream), thrust::make_counting_iterator<size_type>(0), strings_count,
          [d_strings, d_offsets, d_chars] __device__(size_type idx){
              // place individual strings
              auto item = d_strings[idx];
              if( item.first!=nullptr )
                  memcpy(d_chars + d_offsets[idx], item.first, item.second );
          });

    return make_strings_column(strings_count, std::move(offsets_column), std::move(chars_column),
                               null_count, std::move(null_mask), stream, mr);
}

// Create a strings-type column from device vector of chars and vector of offsets.
std::unique_ptr<column> make_strings_column(
    const rmm::device_vector<char>& strings,
    const rmm::device_vector<size_type>& offsets,
    const rmm::device_vector<bitmask_type>& valid_mask,
    size_type null_count,
    hipStream_t stream,
    rmm::mr::device_memory_resource* mr )
{
    size_type num_strings = offsets.size()-1;
    CUDF_EXPECTS( num_strings > 0, "strings count must be greater than 0");
    CUDF_EXPECTS( null_count < num_strings, "null strings column not yet supported");
    if( null_count > 0 ) {
        CUDF_EXPECTS( !valid_mask.empty(), "Cannot have null elements without a null mask." );
    }

    auto execpol = rmm::exec_policy(stream);
    size_type bytes = offsets.back();
    CUDF_EXPECTS( bytes >=0, "invalid offsets vector");

    // build offsets column -- this is the number of strings + 1
    auto offsets_column = make_numeric_column( data_type{INT32}, num_strings+1, mask_state::UNALLOCATED, stream, mr );
    auto offsets_view = offsets_column->mutable_view();
    CUDA_TRY(hipMemcpyAsync( offsets_view.data<int32_t>(), offsets.data().get(),
                              (num_strings+1)*sizeof(int32_t),
                              hipMemcpyDeviceToDevice, stream ));
    // build null bitmask
    rmm::device_buffer null_mask{
        valid_mask.data().get(),
        valid_mask.size() * sizeof(bitmask_type)}; // Or this works too: sizeof(typename std::remove_reference_t<decltype(valid_mask)>::value_type)
                                                   // Following give the incorrect value of 8 instead of 4 because of smart references:
                                                   // sizeof(valid_mask[0]), sizeof(decltype(valid_mask.front()))

    // build chars column
    auto chars_column = strings::detail::create_chars_child_column( num_strings, null_count, bytes, mr, stream );
    auto chars_view = chars_column->mutable_view();
    CUDA_TRY(hipMemcpyAsync( chars_view.data<char>(), strings.data().get(), bytes,
                              hipMemcpyDeviceToDevice, stream ));

    return make_strings_column(num_strings, std::move(offsets_column), std::move(chars_column),
                               null_count, std::move(null_mask), stream, mr);
}

// Create strings column from host vectors
std::unique_ptr<column> make_strings_column(
    const std::vector<char>& strings, const std::vector<size_type>& offsets,
    const std::vector<bitmask_type>& null_mask, size_type null_count,
    hipStream_t stream, rmm::mr::device_memory_resource* mr) {
  rmm::device_vector<char> d_strings{strings};
  rmm::device_vector<size_type> d_offsets{offsets};
  rmm::device_vector<bitmask_type> d_null_mask{null_mask};

  return make_strings_column(d_strings, d_offsets, d_null_mask, null_count,
                             stream, mr);
}

//
std::unique_ptr<column> make_strings_column(
    size_type num_strings,
    std::unique_ptr<column> offsets_column,
    std::unique_ptr<column> chars_column,
    size_type null_count,
    rmm::device_buffer&& null_mask,
    hipStream_t stream,
    rmm::mr::device_memory_resource* mr)
{
    if( null_count > 0 )
        CUDF_EXPECTS( null_mask.size() > 0, "Column with nulls must be nullable.");
    CUDF_EXPECTS( num_strings == offsets_column->size()-1, "Invalid offsets column size for strings column." );
    CUDF_EXPECTS( offsets_column->null_count()==0, "Offsets column should not contain nulls");
    CUDF_EXPECTS( chars_column->null_count()==0, "Chars column should not contain nulls");

    std::vector<std::unique_ptr<column>> children;
    children.emplace_back(std::move(offsets_column));
    children.emplace_back(std::move(chars_column));
    return std::make_unique<column>(
        data_type{STRING}, num_strings, rmm::device_buffer{0,stream,mr},
        null_mask, null_count,
        std::move(children));
}

}  // namespace cudf
