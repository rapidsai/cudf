#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <bitmask/valid_if.cuh>
#include <cudf/column/column_factories.hpp>
#include <cudf/column/column.hpp>
#include <cudf/functions.h>
#include <cudf/null_mask.hpp>
#include <utilities/error_utils.hpp>

#include <rmm/thrust_rmm_allocator.h>
#include <thrust/transform_reduce.h>
#include <thrust/transform_scan.h>
#include <thrust/for_each.h>


namespace cudf {

// Create a strings-type column from array of pointer/size pairs
std::unique_ptr<column> make_strings_column(
    const rmm::device_vector<thrust::pair<const char*,size_type>>& strings,
    hipStream_t stream,
    rmm::mr::device_memory_resource* mr)
{
    size_type num_strings = strings.size();
    // maybe a separate factory for creating null strings-column
    CUDF_EXPECTS(num_strings > 0, "must specify at least one pair");

    auto execpol = rmm::exec_policy(stream);
    auto d_strings = strings.data().get();

    // check total size is not too large for cudf column
    size_t bytes = thrust::transform_reduce( execpol->on(stream),
        thrust::make_counting_iterator<size_t>(0),
        thrust::make_counting_iterator<size_t>(num_strings),
        [d_strings] __device__ (size_t idx) {
            auto item = d_strings[idx];
            return (item.first!=nullptr) ? item.second : 0;
        },
        0, thrust::plus<size_t>());
    CUDF_EXPECTS( bytes < std::numeric_limits<size_type>::max(), "total size of strings is too large for cudf column" );

    // build offsets column -- last entry is the total size
    auto offsets_column = make_numeric_column( data_type{INT32}, num_strings+1, mask_state::UNALLOCATED, stream, mr );
    auto offsets_view = offsets_column->mutable_view();
    auto d_offsets = offsets_view.data<int32_t>();
    // Using inclusive-scan to compute last entry which is the total size.
    // Exclusive-scan is possible but will not compute that last entry.
    // Rather than manually computing the final offset using values in device memory,
    // we use inclusive-scan on a shifted output (d_offsets+1) and then set the first
    // zero offset manually.
    thrust::transform_inclusive_scan( execpol->on(stream),
        thrust::make_counting_iterator<size_type>(0), thrust::make_counting_iterator<size_type>(num_strings),
        d_offsets+1, // fills in offsets entries [1,num_strings]
        [d_strings] __device__ (size_type idx) {
            thrust::pair<const char*,size_type> item = d_strings[idx];
            return ( item.first!=nullptr ? static_cast<int32_t>(item.second) : 0 );
        },
        thrust::plus<int32_t>() );
    // set the first offset to 0
    CUDA_TRY(hipMemsetAsync( d_offsets, 0, sizeof(*d_offsets), stream));

    // create null mask
    auto valid_mask = valid_if( static_cast<const bit_mask_t*>(nullptr),
        [d_strings] __device__ (size_type idx) { return d_strings[idx].first!=nullptr; },
        num_strings, stream );
    auto null_count = valid_mask.second;
    rmm::device_buffer null_mask(valid_mask.first, gdf_valid_allocation_size(num_strings),
                                 stream, mr);
    RMM_TRY( RMM_FREE(valid_mask.first,stream) ); // TODO valid_if to return device_buffer in future
    // if we have all nulls, a null chars column is allowed
    // if all non-null strings are empty strings, we need a non-null chars column
    // - in this case we set the bytes to 1 to create a minimal one-byte chars column
    if( (bytes==0) && (null_count < num_strings) )
        bytes = 1; // all entries are empty strings

    // build chars column
    auto chars_column = make_numeric_column( data_type{INT8}, bytes, mask_state::UNALLOCATED, stream, mr );
    auto chars_view = chars_column->mutable_view();
    auto d_chars = chars_view.data<char>();
    thrust::for_each_n(execpol->on(stream), thrust::make_counting_iterator<size_type>(0), num_strings,
          [d_strings, d_offsets, d_chars] __device__(size_type idx){
              // place individual strings
              auto item = d_strings[idx];
              if( item.first!=nullptr )
                  memcpy(d_chars + d_offsets[idx], item.first, item.second );
          });

    // build children vector
    std::vector<std::unique_ptr<column>> children;
    children.emplace_back(std::move(offsets_column));
    children.emplace_back(std::move(chars_column));

    // no data-ptr with num_strings elements plus children
    return std::make_unique<column>(
        data_type{STRING}, num_strings, rmm::device_buffer{0,stream,mr},
        null_mask, null_count,
        std::move(children));
}

// Create a strings-type column from array of chars and array of offsets.
std::unique_ptr<column> make_strings_column(
    const rmm::device_vector<char>& strings,
    const rmm::device_vector<size_type>& offsets,
    const rmm::device_vector<bitmask_type>& valid_mask,
    size_type null_count,
    hipStream_t stream,
    rmm::mr::device_memory_resource* mr )
{
    size_type num_strings = offsets.size()-1;
    CUDF_EXPECTS( num_strings > 0, "strings count must be greater than 0");
    CUDF_EXPECTS( null_count < num_strings, "null strings column not yet supported");
    if( null_count > 0 ) {
        CUDF_EXPECTS( !valid_mask.empty(), "Cannot have null elements without a null mask." );
    }

    auto execpol = rmm::exec_policy(stream);
    size_type bytes = offsets.back() - offsets[0];
    CUDF_EXPECTS( bytes >=0, "invalid offsets vector");

    // build offsets column -- this is the number of strings + 1
    auto offsets_column = make_numeric_column( data_type{INT32}, num_strings+1, mask_state::UNALLOCATED, stream, mr );
    auto offsets_view = offsets_column->mutable_view();
    CUDA_TRY(hipMemcpyAsync( offsets_view.data<int32_t>(), offsets.data().get(),
                              (num_strings+1)*sizeof(int32_t),
                              hipMemcpyDeviceToDevice, stream ));

    // build null bitmask
    rmm::device_buffer null_mask;
    if( null_count )
        null_mask = rmm::device_buffer(valid_mask.data().get(),
                                       gdf_valid_allocation_size(num_strings),
                                       stream, mr);

    // build chars column
    auto chars_column = make_numeric_column( data_type{INT8}, bytes, mask_state::UNALLOCATED, stream, mr );
    auto chars_view = chars_column->mutable_view();
    CUDA_TRY(hipMemcpyAsync( chars_view.data<char>(), strings.data().get(), bytes,
                              hipMemcpyDeviceToDevice, stream ));

    // build children vector
    std::vector<std::unique_ptr<column>> children;
    children.emplace_back(std::move(offsets_column));
    children.emplace_back(std::move(chars_column));

    //
    return std::make_unique<column>(
        data_type{STRING}, num_strings, rmm::device_buffer{0,stream,mr},
        null_mask, null_count,
        std::move(children));
}

}  // namespace cudf
