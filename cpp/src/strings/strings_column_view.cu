/*
 * Copyright (c) 2019-2021, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <cudf/column/column_device_view.cuh>
#include <cudf/detail/get_value.cuh>
#include <cudf/detail/nvtx/ranges.hpp>
#include <cudf/strings/string_view.cuh>
#include <cudf/strings/strings_column_view.hpp>
#include <cudf/utilities/error.hpp>

#include <rmm/cuda_stream_view.hpp>
#include <rmm/device_uvector.hpp>
#include <rmm/exec_policy.hpp>

#include <thrust/transform.h>

namespace cudf {
//
strings_column_view::strings_column_view(column_view strings_column) : column_view(strings_column)
{
  CUDF_EXPECTS(type().id() == type_id::STRING, "strings_column_view only supports strings");
}

column_view strings_column_view::parent() const { return static_cast<column_view>(*this); }

column_view strings_column_view::offsets() const
{
  CUDF_EXPECTS(num_children() > 0, "strings column has no children");
  return child(offsets_column_index);
}

column_view strings_column_view::chars() const
{
  CUDF_EXPECTS(num_children() > 0, "strings column has no children");
  return child(chars_column_index);
}

size_type strings_column_view::chars_size() const noexcept
{
  if (size() == 0) return 0;
  return chars().size();
}

namespace strings {

std::pair<rmm::device_uvector<char>, rmm::device_uvector<size_type>> create_offsets(
  strings_column_view const& strings,
  rmm::cuda_stream_view stream,
  rmm::mr::device_memory_resource* mr)
{
  CUDF_FUNC_RANGE();
  size_type const count = strings.size();

  auto d_offsets = strings.offsets().data<int32_t>();
  d_offsets += strings.offset();  // nvbug-2808421 : do not combine with the previous line

  rmm::device_uvector<size_type> offsets(count + 1, stream);
  // normalize the offset values for the column offset
  thrust::transform(rmm::exec_policy(stream),
                    d_offsets,
                    d_offsets + count + 1,
                    offsets.begin(),
                    [d_offsets] __device__(int32_t offset) {
                      return static_cast<size_type>(offset - d_offsets[0]);
                    });

  // get the input chars column byte offset
  auto const bytes = offsets.element(count, stream);
  auto const chars_offset =
    cudf::detail::get_value<offset_type>(strings.offsets(), strings.offset(), stream);
  stream.synchronize();

  // copy the chars column data
  const char* d_chars = strings.chars().data<char>() + chars_offset;
  rmm::device_uvector<char> chars(bytes, stream);
  CUDA_TRY(hipMemcpyAsync(chars.data(), d_chars, bytes, hipMemcpyDefault, stream.value()));

  // return offsets and chars
  return std::make_pair(std::move(chars), std::move(offsets));
}

}  // namespace strings
}  // namespace cudf
