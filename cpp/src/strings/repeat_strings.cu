#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2021, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <cudf/column/column_device_view.cuh>
#include <cudf/column/column_factories.hpp>
#include <cudf/detail/null_mask.hpp>
#include <cudf/detail/nvtx/ranges.hpp>
#include <cudf/strings/detail/utilities.cuh>
#include <cudf/strings/detail/utilities.hpp>
#include <cudf/strings/repeat_strings.hpp>
#include <cudf/strings/strings_column_view.hpp>
#include <cudf/utilities/error.hpp>

#include <rmm/cuda_stream_view.hpp>

#include <thrust/transform.h>

namespace cudf {
namespace strings {
namespace detail {

std::unique_ptr<string_scalar> repeat_strings(string_scalar const& input,
                                              size_type repeat_times,
                                              rmm::cuda_stream_view stream,
                                              rmm::mr::device_memory_resource* mr)
{
  if (!input.is_valid(stream)) { return std::make_unique<string_scalar>("", false, stream, mr); }
  if (input.size() == 0 || repeat_times <= 0) {
    return std::make_unique<string_scalar>("", true, stream, mr);
  }
  if (repeat_times == 1) { return std::make_unique<string_scalar>(input, stream, mr); }

  CUDF_EXPECTS(input.size() <= std::numeric_limits<size_type>::max() / repeat_times,
               "The output string has size that exceeds the maximum allowed size.");

  auto const str_size = input.size();
  auto const iter     = thrust::make_counting_iterator(0);
  auto buff           = rmm::device_buffer(repeat_times * input.size(), stream, mr);

  // Pull data from the input string into each byte of the output string.
  thrust::transform(rmm::exec_policy(stream),
                    iter,
                    iter + repeat_times * str_size,
                    static_cast<char*>(buff.data()),
                    [in_ptr = input.data(), str_size] __device__(const auto idx) {
                      return in_ptr[idx % str_size];
                    });

  return std::make_unique<string_scalar>(std::move(buff));
}

namespace {
/**
 * @brief Generate a strings column in which each row is an empty or null string.
 *
 * The output strings column has the same bitmask as the input column.
 */
auto generate_empty_output(strings_column_view const& input,
                           size_type strings_count,
                           rmm::cuda_stream_view stream,
                           rmm::mr::device_memory_resource* mr)
{
  auto chars_column = create_chars_child_column(0, stream, mr);

  auto offsets_column = make_numeric_column(
    data_type{type_to_id<offset_type>()}, strings_count + 1, mask_state::UNALLOCATED, stream, mr);
  CUDA_TRY(hipMemsetAsync(offsets_column->mutable_view().template data<offset_type>(),
                           0,
                           offsets_column->size() * sizeof(offset_type),
                           stream.value()));

  return make_strings_column(strings_count,
                             std::move(offsets_column),
                             std::move(chars_column),
                             input.null_count(),
                             cudf::detail::copy_bitmask(input.parent(), stream, mr),
                             stream,
                             mr);
}

/**
 * @brief Functor to compute string sizes and repeat the input strings.
 *
 * This functor is called only when `repeat_times > 0`. In addition, the total number of threads
 * running this functor is `repeat_times * strings_count` (instead of `string_count`) for maximizing
 * parallelism and better load-balancing.
 */
struct compute_size_and_repeat_fn {
  column_device_view const strings_dv;
  size_type const repeat_times;
  bool const has_nulls;

  offset_type* d_offsets{nullptr};

  // If d_chars == nullptr: only compute sizes of the output strings.
  // If d_chars != nullptr: only repeat strings.
  char* d_chars{nullptr};

  // `idx` will be in the range of [0, repeat_times * strings_count).
  __device__ void operator()(size_type const idx) const noexcept
  {
    auto const str_idx    = idx / repeat_times;  // value cycles in [0, string_count)
    auto const repeat_idx = idx % repeat_times;  // value cycles in [0, repeat_times)
    auto const is_valid   = !has_nulls || strings_dv.is_valid_nocheck(str_idx);

    if (!d_chars && repeat_idx == 0) {
      d_offsets[str_idx] =
        is_valid ? repeat_times * strings_dv.element<string_view>(str_idx).size_bytes() : 0;
    }

    // Each input string will be copied by `repeat_times` threads into the output string.
    if (d_chars && is_valid) {
      auto const d_str    = strings_dv.element<string_view>(str_idx);
      auto const str_size = d_str.size_bytes();
      if (str_size > 0) {
        auto const input_ptr  = d_str.data();
        auto const output_ptr = d_chars + d_offsets[str_idx] + repeat_idx * str_size;
        std::memcpy(output_ptr, input_ptr, str_size);
      }
    }
  }
};

}  // namespace

std::unique_ptr<column> repeat_strings(strings_column_view const& input,
                                       size_type repeat_times,
                                       rmm::cuda_stream_view stream,
                                       rmm::mr::device_memory_resource* mr)
{
  auto const strings_count = input.size();
  if (strings_count == 0) { return make_empty_column(data_type{type_id::STRING}); }

  if (repeat_times <= 0) {
    // If the number of repetitions is not positive, each row of the output strings column will be
    // either an empty string (if the input row is not null), or a null (if the input row is null).
    return generate_empty_output(input, strings_count, stream, mr);
  }

  // If `repeat_times == 1`, just make a copy of the input.
  if (repeat_times == 1) { return std::make_unique<column>(input.parent(), stream, mr); }

  auto const strings_dv_ptr = column_device_view::create(input.parent(), stream);
  auto const fn = compute_size_and_repeat_fn{*strings_dv_ptr, repeat_times, input.has_nulls()};

  // Repeat the strings in each row.
  // Note that this cannot handle the cases when the size of the output column exceeds the maximum
  // value that can be indexed by size_type (offset_type).
  // In such situations, an exception may be thrown, or the output result is undefined.
  auto [offsets_column, chars_column] =
    make_strings_children(fn, strings_count * repeat_times, strings_count, stream, mr);

  return make_strings_column(strings_count,
                             std::move(offsets_column),
                             std::move(chars_column),
                             input.null_count(),
                             cudf::detail::copy_bitmask(input.parent(), stream, mr),
                             stream,
                             mr);
}

}  // namespace detail

std::unique_ptr<string_scalar> repeat_strings(string_scalar const& input,
                                              size_type repeat_times,
                                              rmm::mr::device_memory_resource* mr)
{
  CUDF_FUNC_RANGE();
  return detail::repeat_strings(input, repeat_times, rmm::cuda_stream_default, mr);
}

std::unique_ptr<column> repeat_strings(strings_column_view const& input,
                                       size_type repeat_times,
                                       rmm::mr::device_memory_resource* mr)
{
  CUDF_FUNC_RANGE();
  return detail::repeat_strings(input, repeat_times, rmm::cuda_stream_default, mr);
}

}  // namespace strings
}  // namespace cudf
