#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2021, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <cudf/column/column_device_view.cuh>
#include <cudf/column/column_factories.hpp>
#include <cudf/detail/indexalator.cuh>
#include <cudf/detail/null_mask.hpp>
#include <cudf/detail/nvtx/ranges.hpp>
#include <cudf/strings/detail/utilities.cuh>
#include <cudf/strings/repeat_strings.hpp>
#include <cudf/strings/strings_column_view.hpp>
#include <cudf/utilities/error.hpp>
#include <cudf/utilities/type_dispatcher.hpp>

#include <rmm/cuda_stream_view.hpp>

#include <thrust/functional.h>
#include <thrust/transform.h>
#include <thrust/transform_reduce.h>

namespace cudf {
namespace strings {
namespace detail {
std::unique_ptr<string_scalar> repeat_string(string_scalar const& input,
                                             size_type repeat_times,
                                             rmm::cuda_stream_view stream,
                                             rmm::mr::device_memory_resource* mr)
{
  if (!input.is_valid(stream)) { return std::make_unique<string_scalar>("", false, stream, mr); }
  if (input.size() == 0 || repeat_times <= 0) {
    return std::make_unique<string_scalar>("", true, stream, mr);
  }
  if (repeat_times == 1) { return std::make_unique<string_scalar>(input, stream, mr); }

  CUDF_EXPECTS(input.size() <= std::numeric_limits<size_type>::max() / repeat_times,
               "The output string has size that exceeds the maximum allowed size.");

  auto const str_size = input.size();
  auto const iter     = thrust::make_counting_iterator(0);
  auto buff           = rmm::device_buffer(repeat_times * input.size(), stream, mr);

  // Pull data from the input string into each byte of the output string.
  thrust::transform(rmm::exec_policy(stream),
                    iter,
                    iter + repeat_times * str_size,
                    static_cast<char*>(buff.data()),
                    [in_ptr = input.data(), str_size] __device__(const auto idx) {
                      return in_ptr[idx % str_size];
                    });

  return std::make_unique<string_scalar>(std::move(buff));
}

namespace {
/**
 * @brief Generate a strings column in which each row is an empty string or a null.
 *
 * The output strings column has the same bitmask as the input column.
 */
auto generate_empty_output(strings_column_view const& input,
                           size_type strings_count,
                           rmm::cuda_stream_view stream,
                           rmm::mr::device_memory_resource* mr)
{
  auto chars_column = create_chars_child_column(0, stream, mr);

  auto offsets_column = make_numeric_column(
    data_type{type_to_id<offset_type>()}, strings_count + 1, mask_state::UNALLOCATED, stream, mr);
  CUDA_TRY(hipMemsetAsync(offsets_column->mutable_view().template data<offset_type>(),
                           0,
                           offsets_column->size() * sizeof(offset_type),
                           stream.value()));

  return make_strings_column(strings_count,
                             std::move(offsets_column),
                             std::move(chars_column),
                             input.null_count(),
                             cudf::detail::copy_bitmask(input.parent(), stream, mr),
                             stream,
                             mr);
}

/**
 * @brief Functor to compute output string sizes and repeat the input strings.
 *
 * This functor is called only when `repeat_times > 0`. In addition, the total number of threads
 * running this functor is `repeat_times * strings_count` (instead of `string_count`) for maximizing
 * parallelism and better load-balancing.
 */
struct compute_size_and_repeat_fn {
  column_device_view const strings_dv;
  size_type const repeat_times;
  bool const has_nulls;

  offset_type* d_offsets{nullptr};

  // If d_chars == nullptr: only compute sizes of the output strings.
  // If d_chars != nullptr: only repeat strings.
  char* d_chars{nullptr};

  // `idx` will be in the range of [0, repeat_times * strings_count).
  __device__ void operator()(size_type const idx) const noexcept
  {
    auto const str_idx    = idx / repeat_times;  // value cycles in [0, string_count)
    auto const repeat_idx = idx % repeat_times;  // value cycles in [0, repeat_times)
    auto const is_valid   = !has_nulls || strings_dv.is_valid_nocheck(str_idx);

    if (!d_chars && repeat_idx == 0) {
      d_offsets[str_idx] =
        is_valid ? repeat_times * strings_dv.element<string_view>(str_idx).size_bytes() : 0;
    }

    // Each input string will be copied by `repeat_times` threads into the output string.
    if (d_chars && is_valid) {
      auto const d_str    = strings_dv.element<string_view>(str_idx);
      auto const str_size = d_str.size_bytes();
      if (str_size > 0) {
        auto const input_ptr  = d_str.data();
        auto const output_ptr = d_chars + d_offsets[str_idx] + repeat_idx * str_size;
        std::memcpy(output_ptr, input_ptr, str_size);
      }
    }
  }
};

}  // namespace

std::unique_ptr<column> repeat_strings(strings_column_view const& input,
                                       size_type repeat_times,
                                       rmm::cuda_stream_view stream,
                                       rmm::mr::device_memory_resource* mr)
{
  auto const strings_count = input.size();
  if (strings_count == 0) { return make_empty_column(data_type{type_id::STRING}); }

  if (repeat_times <= 0) {
    // If the number of repetitions is not positive, each row of the output strings column will be
    // either an empty string (if the input row is not null), or a null (if the input row is null).
    return generate_empty_output(input, strings_count, stream, mr);
  }

  // If `repeat_times == 1`, just make a copy of the input.
  if (repeat_times == 1) { return std::make_unique<column>(input.parent(), stream, mr); }

  auto const strings_dv_ptr = column_device_view::create(input.parent(), stream);
  auto const fn = compute_size_and_repeat_fn{*strings_dv_ptr, repeat_times, input.has_nulls()};

  auto [offsets_column, chars_column] =
    make_strings_children(fn, strings_count * repeat_times, strings_count, stream, mr);
  return make_strings_column(strings_count,
                             std::move(offsets_column),
                             std::move(chars_column),
                             input.null_count(),
                             cudf::detail::copy_bitmask(input.parent(), stream, mr),
                             stream,
                             mr);
}

namespace {
/**
 * @brief Functor to compute string sizes and repeat the input strings, each string is repeated by a
 * separate number of times.
 */
template <class Iterator>
struct compute_size_and_repeat_separately_fn {
  column_device_view const strings_dv;
  column_device_view const repeat_times_dv;
  Iterator const repeat_times_iter;
  bool const strings_has_nulls;
  bool const rtimes_has_nulls;

  offset_type* d_offsets{nullptr};

  // If d_chars == nullptr: only compute sizes of the output strings.
  // If d_chars != nullptr: only repeat strings.
  char* d_chars{nullptr};

  __device__ int64_t operator()(size_type const idx) const noexcept
  {
    auto const string_is_valid = !strings_has_nulls || strings_dv.is_valid_nocheck(idx);
    auto const rtimes_is_valid = !rtimes_has_nulls || repeat_times_dv.is_valid_nocheck(idx);

    // Any null input (either string or repeat_times value) will result in a null output.
    auto const is_valid = string_is_valid && rtimes_is_valid;

    // When the input string is null, `repeat_times` and `string_size` are also set to 0.
    // This makes sure that if `repeat_times > 0` then we will always have a valid input string,
    // and if `repeat_times <= 0` we will never copy anything to the output.
    auto const repeat_times = is_valid ? repeat_times_iter[idx] : size_type{0};
    auto const string_size =
      is_valid ? strings_dv.element<string_view>(idx).size_bytes() : size_type{0};

    // The output_size is returned, and it needs to be an int64_t number to prevent overflow.
    auto const output_size =
      repeat_times > 0 ? static_cast<int64_t>(repeat_times) * static_cast<int64_t>(string_size)
                       : int64_t{0};

    if (!d_chars) {
      // If overflow happen, the stored value of output string size will be incorrect due to
      // downcasting. In such cases, the entire output string size array should be discarded.
      d_offsets[idx] = static_cast<size_type>(output_size);
    } else if (repeat_times > 0) {
      auto const d_str    = strings_dv.element<string_view>(idx);
      auto const str_size = d_str.size_bytes();
      if (str_size > 0) {
        auto const input_ptr = d_str.data();
        auto output_ptr      = d_chars + d_offsets[idx];
        for (size_type repeat_idx = 0; repeat_idx < repeat_times; ++repeat_idx) {
          output_ptr = copy_and_increment(output_ptr, input_ptr, str_size);
        }
      }
    }

    // The output_size value may be used to sum up to detect overflow at the caller site.
    // The caller can detect overflow easily by checking `SUM(output_size) > INT_MAX`.
    return output_size;
  }
};

/**
 * @brief Creates child offsets and chars columns by applying the template function that
 * can be used for computing the output size of each string as well as create the output.
 *
 * This function is similar to `strings::detail::make_strings_children`, except that it accepts an
 * optional input `std::optional<column_view>` that can contain the precomputed sizes of the output
 * strings.
 */
template <typename Func>
auto make_strings_children(Func fn,
                           size_type exec_size,
                           size_type strings_count,
                           std::optional<column_view> output_strings_sizes,
                           rmm::cuda_stream_view stream,
                           rmm::mr::device_memory_resource* mr)
{
  auto offsets_column = make_numeric_column(
    data_type{type_id::INT32}, strings_count + 1, mask_state::UNALLOCATED, stream, mr);

  auto offsets_view = offsets_column->mutable_view();
  auto d_offsets    = offsets_view.template data<size_type>();
  fn.d_offsets      = d_offsets;

  // This may be called twice -- once for offsets and once for chars.
  auto for_each_fn = [exec_size, stream](Func& fn) {
    thrust::for_each_n(
      rmm::exec_policy(stream), thrust::make_counting_iterator<size_type>(0), exec_size, fn);
  };

  if (!output_strings_sizes.has_value()) {
    // Compute the output sizes only if they are not given.
    for_each_fn(fn);

    // Compute the offsets values.
    thrust::exclusive_scan(
      rmm::exec_policy(stream), d_offsets, d_offsets + strings_count + 1, d_offsets);
  } else {
    // Compute the offsets values from the provided output string sizes.
    auto const string_sizes = output_strings_sizes.value();
    CUDA_TRY(hipMemsetAsync(d_offsets, 0, sizeof(offset_type), stream.value()));
    thrust::inclusive_scan(rmm::exec_policy(stream),
                           string_sizes.template begin<size_type>(),
                           string_sizes.template end<size_type>(),
                           d_offsets + 1);
  }

  // Now build the chars column
  auto const bytes  = cudf::detail::get_value<size_type>(offsets_view, strings_count, stream);
  auto chars_column = create_chars_child_column(bytes, stream, mr);

  // Execute the function fn again to fill the chars column.
  // Note that if the output chars column has zero size, the function fn should not be called to
  // avoid accidentally overwriting the offsets.
  if (bytes > 0) {
    fn.d_chars = chars_column->mutable_view().template data<char>();
    for_each_fn(fn);
  }

  return std::make_pair(std::move(offsets_column), std::move(chars_column));
}

}  // namespace

std::unique_ptr<column> repeat_strings(strings_column_view const& input,
                                       column_view const& repeat_times,
                                       std::optional<column_view> output_strings_sizes,
                                       rmm::cuda_stream_view stream,
                                       rmm::mr::device_memory_resource* mr)
{
  CUDF_EXPECTS(input.size() == repeat_times.size(), "The input columns must have the same size.");
  CUDF_EXPECTS(cudf::is_index_type(repeat_times.type()),
               "repeat_strings expects an integer type for the `repeat_times` input column.");
  if (output_strings_sizes.has_value()) {
    auto const output_sizes = output_strings_sizes.value();
    CUDF_EXPECTS(input.size() == output_sizes.size() &&
                   (!output_sizes.nullable() || !output_sizes.has_nulls()),
                 "The given column of output string sizes is invalid.");
  }

  auto const strings_count = input.size();
  if (strings_count == 0) { return make_empty_column(data_type{type_id::STRING}); }

  auto const strings_dv_ptr      = column_device_view::create(input.parent(), stream);
  auto const repeat_times_dv_ptr = column_device_view::create(repeat_times, stream);
  auto const strings_has_nulls   = input.has_nulls();
  auto const rtimes_has_nulls    = repeat_times.has_nulls();
  auto const repeat_times_iter =
    cudf::detail::indexalator_factory::make_input_iterator(repeat_times);
  auto const fn = compute_size_and_repeat_separately_fn<decltype(repeat_times_iter)>{
    *strings_dv_ptr, *repeat_times_dv_ptr, repeat_times_iter, strings_has_nulls, rtimes_has_nulls};

  auto [offsets_column, chars_column] =
    make_strings_children(fn, strings_count, strings_count, output_strings_sizes, stream, mr);

  // If only one input column has nulls, we just copy its null mask and null count.
  // If both input columns have nulls, we generate new bitmask by AND their bitmasks.
  auto [null_mask, null_count] = [&] {
    if (strings_has_nulls ^ rtimes_has_nulls) {
      auto const& col = strings_has_nulls ? input.parent() : repeat_times;
      return std::make_pair(cudf::detail::copy_bitmask(col, stream, mr), col.null_count());
    } else if (strings_has_nulls && rtimes_has_nulls) {
      return std::make_pair(
        cudf::detail::bitmask_and(table_view{{input.parent(), repeat_times}}, stream, mr),
        UNKNOWN_NULL_COUNT);
    }
    return std::make_pair(rmm::device_buffer{0, stream, mr}, 0);
  }();

  return make_strings_column(strings_count,
                             std::move(offsets_column),
                             std::move(chars_column),
                             null_count,
                             std::move(null_mask),
                             stream,
                             mr);
}

std::pair<std::unique_ptr<column>, int64_t> repeat_strings_output_sizes(
  strings_column_view const& input,
  column_view const& repeat_times,
  rmm::cuda_stream_view stream,
  rmm::mr::device_memory_resource* mr)
{
  CUDF_EXPECTS(input.size() == repeat_times.size(), "The input columns must have the same size.");
  CUDF_EXPECTS(
    cudf::is_index_type(repeat_times.type()),
    "repeat_strings_output_sizes expects an integer type for the `repeat_times` input column.");

  auto const strings_count = input.size();
  if (strings_count == 0) {
    return std::make_pair(make_empty_column(data_type{type_to_id<size_type>()}), int64_t{0});
  }

  auto output_sizes = make_numeric_column(
    data_type{type_to_id<size_type>()}, strings_count, mask_state::UNALLOCATED, stream, mr);

  auto const strings_dv_ptr      = column_device_view::create(input.parent(), stream);
  auto const repeat_times_dv_ptr = column_device_view::create(repeat_times, stream);
  auto const strings_has_nulls   = input.has_nulls();
  auto const rtimes_has_nulls    = repeat_times.has_nulls();
  auto const repeat_times_iter =
    cudf::detail::indexalator_factory::make_input_iterator(repeat_times);

  auto const fn = compute_size_and_repeat_separately_fn<decltype(repeat_times_iter)>{
    *strings_dv_ptr,
    *repeat_times_dv_ptr,
    repeat_times_iter,
    strings_has_nulls,
    rtimes_has_nulls,
    output_sizes->mutable_view().template begin<size_type>()};

  auto const total_bytes =
    thrust::transform_reduce(rmm::exec_policy(stream),
                             thrust::make_counting_iterator<size_type>(0),
                             thrust::make_counting_iterator<size_type>(strings_count),
                             fn,
                             int64_t{0},
                             thrust::plus<int64_t>{});

  return std::make_pair(std::move(output_sizes), total_bytes);
}

}  // namespace detail

std::unique_ptr<string_scalar> repeat_string(string_scalar const& input,
                                             size_type repeat_times,
                                             rmm::mr::device_memory_resource* mr)
{
  CUDF_FUNC_RANGE();
  return detail::repeat_string(input, repeat_times, rmm::cuda_stream_default, mr);
}

std::unique_ptr<column> repeat_strings(strings_column_view const& input,
                                       size_type repeat_times,
                                       rmm::mr::device_memory_resource* mr)
{
  CUDF_FUNC_RANGE();
  return detail::repeat_strings(input, repeat_times, rmm::cuda_stream_default, mr);
}

std::unique_ptr<column> repeat_strings(strings_column_view const& input,
                                       column_view const& repeat_times,
                                       std::optional<column_view> output_strings_sizes,
                                       rmm::mr::device_memory_resource* mr)
{
  CUDF_FUNC_RANGE();
  return detail::repeat_strings(
    input, repeat_times, output_strings_sizes, rmm::cuda_stream_default, mr);
}

std::pair<std::unique_ptr<column>, int64_t> repeat_strings_output_sizes(
  strings_column_view const& input,
  column_view const& repeat_times,
  rmm::mr::device_memory_resource* mr)
{
  CUDF_FUNC_RANGE();
  return detail::repeat_strings_output_sizes(input, repeat_times, rmm::cuda_stream_default, mr);
}

}  // namespace strings
}  // namespace cudf
