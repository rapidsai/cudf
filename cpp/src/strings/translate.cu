/*
 * Copyright (c) 2019-2020, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <strings/utilities.cuh>

#include <cudf/column/column.hpp>
#include <cudf/column/column_device_view.cuh>
#include <cudf/column/column_factories.hpp>
#include <cudf/detail/null_mask.hpp>
#include <cudf/detail/nvtx/ranges.hpp>
#include <cudf/strings/detail/utilities.hpp>
#include <cudf/strings/string_view.cuh>
#include <cudf/strings/strings_column_view.hpp>
#include <cudf/strings/translate.hpp>

#include <rmm/cuda_stream_view.hpp>

#include <thrust/find.h>

#include <algorithm>

namespace cudf {
namespace strings {
namespace detail {
using translate_table = thrust::pair<char_utf8, char_utf8>;

namespace {
/**
 * @brief This is the translate functor for replacing individual characters
 * in each string.
 */
struct translate_fn {
  column_device_view const d_strings;
  rmm::device_vector<translate_table>::iterator table_begin;
  rmm::device_vector<translate_table>::iterator table_end;
  int32_t const* d_offsets{};
  char* d_chars{};

  __device__ size_type operator()(size_type idx)
  {
    if (d_strings.is_null(idx)) return 0;
    string_view d_str = d_strings.element<string_view>(idx);
    size_type bytes   = d_str.size_bytes();
    char* out_ptr     = d_offsets ? d_chars + d_offsets[idx] : nullptr;
    for (auto chr : d_str) {
      auto entry =
        thrust::find_if(thrust::seq, table_begin, table_end, [chr] __device__(auto const& te) {
          return te.first == chr;
        });
      if (entry != table_end) {
        bytes -= bytes_in_char_utf8(chr);
        chr = static_cast<translate_table>(*entry).second;
        if (chr)  // if null, skip the character
          bytes += bytes_in_char_utf8(chr);
      }
      if (chr && out_ptr) out_ptr += from_char_utf8(chr, out_ptr);
    }
    return bytes;
  }
};

}  // namespace

//
std::unique_ptr<column> translate(
  strings_column_view const& strings,
  std::vector<std::pair<char_utf8, char_utf8>> const& chars_table,
  rmm::cuda_stream_view stream,
  rmm::mr::device_memory_resource* mr = rmm::mr::get_current_device_resource())
{
  size_type strings_count = strings.size();
  if (strings_count == 0) return make_empty_strings_column(stream, mr);

  size_type table_size = static_cast<size_type>(chars_table.size());
  // convert input table
  thrust::host_vector<translate_table> htable(table_size);
  std::transform(chars_table.begin(), chars_table.end(), htable.begin(), [](auto entry) {
    return translate_table{entry.first, entry.second};
  });
  // copy translate table to device memory
  rmm::device_vector<translate_table> table(htable);

  auto execpol        = rmm::exec_policy(stream);
  auto strings_column = column_device_view::create(strings.parent(), stream);
  auto d_strings      = *strings_column;
  // create null mask
  rmm::device_buffer null_mask = cudf::detail::copy_bitmask(strings.parent(), stream, mr);
  // create offsets column
  auto offsets_transformer_itr =
    thrust::make_transform_iterator(thrust::make_counting_iterator<int32_t>(0),
                                    translate_fn{d_strings, table.begin(), table.end()});
  auto offsets_column = make_offsets_child_column(
    offsets_transformer_itr, offsets_transformer_itr + strings_count, stream, mr);
  auto d_offsets = offsets_column->view().data<int32_t>();

  // build chars column
  size_type bytes   = thrust::device_pointer_cast(d_offsets)[strings_count];
  auto chars_column = strings::detail::create_chars_child_column(
    strings_count, strings.null_count(), bytes, stream, mr);
  auto d_chars = chars_column->mutable_view().data<char>();
  thrust::for_each_n(rmm::exec_policy(stream)->on(stream.value()),
                     thrust::make_counting_iterator<cudf::size_type>(0),
                     strings_count,
                     translate_fn{d_strings, table.begin(), table.end(), d_offsets, d_chars});

  return make_strings_column(strings_count,
                             std::move(offsets_column),
                             std::move(chars_column),
                             strings.null_count(),
                             std::move(null_mask),
                             stream,
                             mr);
}

}  // namespace detail

// external APIs

std::unique_ptr<column> translate(strings_column_view const& strings,
                                  std::vector<std::pair<uint32_t, uint32_t>> const& chars_table,
                                  rmm::mr::device_memory_resource* mr)
{
  CUDF_FUNC_RANGE();
  return detail::translate(strings, chars_table, rmm::cuda_stream_default, mr);
}

}  // namespace strings
}  // namespace cudf
