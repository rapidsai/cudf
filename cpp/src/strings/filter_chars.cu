/*
 * Copyright (c) 2020-2021, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <cudf/column/column.hpp>
#include <cudf/column/column_device_view.cuh>
#include <cudf/column/column_factories.hpp>
#include <cudf/detail/get_value.cuh>
#include <cudf/detail/iterator.cuh>
#include <cudf/detail/null_mask.hpp>
#include <cudf/detail/nvtx/ranges.hpp>
#include <cudf/strings/detail/utilities.hpp>
#include <cudf/strings/string_view.cuh>
#include <cudf/strings/strings_column_view.hpp>
#include <cudf/strings/translate.hpp>

#include <strings/utilities.cuh>

#include <rmm/cuda_stream_view.hpp>
#include <rmm/exec_policy.hpp>

#include <thrust/find.h>

#include <algorithm>

namespace cudf {
namespace strings {
namespace detail {

using char_range = thrust::pair<char_utf8, char_utf8>;

namespace {
/**
 * @brief This is the filter functor for replacing characters
 * in each string given a vector of char_range values.
 */
struct filter_fn {
  column_device_view const d_strings;
  filter_type keep_characters;
  rmm::device_uvector<char_range>::iterator table_begin;
  rmm::device_uvector<char_range>::iterator table_end;
  string_view const d_replacement;
  int32_t* d_offsets{};
  char* d_chars{};

  /**
   * @brief Return true if this character should be removed.
   *
   * @param ch Character to check
   * @return True if character should be removed.
   */
  __device__ bool remove_char(char_utf8 ch)
  {
    auto const entry =
      thrust::find_if(thrust::seq, table_begin, table_end, [ch] __device__(auto const& range) {
        return (range.first <= ch) && (ch <= range.second);
      });
    // if keep==true and entry-not-found OR
    // if keep==false and entry-found
    return (keep_characters == filter_type::KEEP) == (entry == table_end);
  }

  /**
   * @brief Execute the filter operation on each string.
   *
   * This is also used to calculate the size of the output.
   *
   * @param idx Index of the current string to process.
   */
  __device__ void operator()(size_type idx)
  {
    if (d_strings.is_null(idx)) {
      if (!d_chars) d_offsets[idx] = 0;
      return;
    }
    auto const d_str = d_strings.element<string_view>(idx);

    auto nbytes  = d_str.size_bytes();
    auto out_ptr = d_chars ? d_chars + d_offsets[idx] : nullptr;
    for (auto itr = d_str.begin(); itr != d_str.end(); ++itr) {
      auto const char_size        = bytes_in_char_utf8(*itr);
      string_view const d_newchar = remove_char(*itr)
                                      ? d_replacement
                                      : string_view(d_str.data() + itr.byte_offset(), char_size);
      if (out_ptr)
        out_ptr = cudf::strings::detail::copy_string(out_ptr, d_newchar);
      else
        nbytes += d_newchar.size_bytes() - char_size;
    }
    if (!out_ptr) d_offsets[idx] = nbytes;
  }
};

}  // namespace

/**
 * @copydoc cudf::strings::filter_characters
 */
std::unique_ptr<column> filter_characters(
  strings_column_view const& strings,
  std::vector<std::pair<cudf::char_utf8, cudf::char_utf8>> characters_to_filter,
  filter_type keep_characters,
  string_scalar const& replacement,
  rmm::cuda_stream_view stream,
  rmm::mr::device_memory_resource* mr)
{
  size_type strings_count = strings.size();
  if (strings_count == 0) return make_empty_strings_column(stream, mr);
  CUDF_EXPECTS(replacement.is_valid(), "Parameter replacement must be valid");
  cudf::string_view d_replacement(replacement.data(), replacement.size());

  // convert input table for copy to device memory
  size_type table_size = static_cast<size_type>(characters_to_filter.size());
  thrust::host_vector<char_range> htable(table_size);
  std::transform(
    characters_to_filter.begin(), characters_to_filter.end(), htable.begin(), [](auto entry) {
      return char_range{entry.first, entry.second};
    });
  rmm::device_uvector<char_range> table(table_size, stream);
  CUDA_TRY(hipMemcpyAsync(table.data(),
                           htable.data(),
                           table_size * sizeof(char_range),
                           hipMemcpyHostToDevice,
                           stream.value()));

  auto d_strings = column_device_view::create(strings.parent(), stream);

  // this utility calls the strip_fn to build the offsets and chars columns
  filter_fn ffn{*d_strings, keep_characters, table.begin(), table.end(), d_replacement};
  auto children = cudf::strings::detail::make_strings_children(
    ffn, strings.size(), strings.null_count(), stream, mr);

  return make_strings_column(strings_count,
                             std::move(children.first),
                             std::move(children.second),
                             strings.null_count(),
                             cudf::detail::copy_bitmask(strings.parent(), stream, mr),
                             stream,
                             mr);
}

}  // namespace detail

/**
 * @copydoc cudf::strings::filter_characters
 */
std::unique_ptr<column> filter_characters(
  strings_column_view const& strings,
  std::vector<std::pair<cudf::char_utf8, cudf::char_utf8>> characters_to_filter,
  filter_type keep_characters,
  string_scalar const& replacement,
  rmm::mr::device_memory_resource* mr)
{
  CUDF_FUNC_RANGE();
  return detail::filter_characters(
    strings, characters_to_filter, keep_characters, replacement, rmm::cuda_stream_default, mr);
}

}  // namespace strings
}  // namespace cudf
