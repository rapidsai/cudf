/*
 * Copyright (c) 2020, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <cudf/column/column.hpp>
#include <cudf/column/column_device_view.cuh>
#include <cudf/column/column_factories.hpp>
#include <cudf/detail/get_value.cuh>
#include <cudf/detail/nvtx/ranges.hpp>
#include <cudf/strings/detail/utilities.hpp>
#include <cudf/strings/string_view.cuh>
#include <cudf/strings/strings_column_view.hpp>
#include <cudf/strings/translate.hpp>
#include <strings/utilities.cuh>

#include <thrust/find.h>
#include <algorithm>

namespace cudf {
namespace strings {
namespace detail {

using char_range = thrust::pair<char_utf8, char_utf8>;

namespace {
/**
 * @brief This is the filter functor for replacing characters
 * in each string given a vector of char_range values.
 */
struct filter_fn {
  column_device_view const d_strings;
  filter_type keep_characters;
  rmm::device_vector<char_range>::iterator table_begin;
  rmm::device_vector<char_range>::iterator table_end;
  string_view const d_replacement;
  int32_t const* d_offsets{};
  char* d_chars{};

  /**
   * @brief Return true if this character should be removed.
   *
   * @param ch Character to check
   * @return True if character should be removed.
   */
  __device__ bool remove_char(char_utf8 ch)
  {
    auto const entry =
      thrust::find_if(thrust::seq, table_begin, table_end, [ch] __device__(auto const& range) {
        return (range.first <= ch) && (ch <= range.second);
      });
    // if keep==true and entry-not-found OR
    // if keep==false and entry-found
    return (keep_characters == filter_type::KEEP) == (entry == table_end);
  }

  /**
   * @brief Execute the filter operation on each string.
   *
   * This is also used to calculate the size of the output.
   *
   * @param idx Index of the current string to process.
   * @return The size of the output for this string.
   */
  __device__ size_type operator()(size_type idx)
  {
    if (d_strings.is_null(idx)) return 0;
    string_view d_str = d_strings.element<string_view>(idx);
    size_type nbytes  = d_str.size_bytes();
    auto const in_ptr = d_str.data();
    auto out_ptr      = d_chars ? d_chars + d_offsets[idx] : nullptr;
    for (auto itr = d_str.begin(); itr != d_str.end(); ++itr) {
      auto const char_size = bytes_in_char_utf8(*itr);
      string_view const d_newchar =
        remove_char(*itr) ? d_replacement : string_view(in_ptr + itr.byte_offset(), char_size);
      nbytes += d_newchar.size_bytes() - char_size;
      if (out_ptr) out_ptr = cudf::strings::detail::copy_string(out_ptr, d_newchar);
    }
    return nbytes;
  }
};

}  // namespace

/**
 * @copydoc cudf::strings::filter_characters
 */
std::unique_ptr<column> filter_characters(
  strings_column_view const& strings,
  std::vector<std::pair<cudf::char_utf8, cudf::char_utf8>> characters_to_filter,
  filter_type keep_characters,
  string_scalar const& replacement,
  hipStream_t stream,
  rmm::mr::device_memory_resource* mr)
{
  size_type strings_count = strings.size();
  if (strings_count == 0) return make_empty_strings_column(mr, stream);
  CUDF_EXPECTS(replacement.is_valid(), "Parameter replacement must be valid");
  cudf::string_view d_replacement(replacement.data(), replacement.size());

  // convert input table for copy to device memory
  size_type table_size = static_cast<size_type>(characters_to_filter.size());
  thrust::host_vector<char_range> htable(table_size);
  std::transform(
    characters_to_filter.begin(), characters_to_filter.end(), htable.begin(), [](auto entry) {
      return char_range{entry.first, entry.second};
    });
  rmm::device_vector<char_range> table(htable);  // copy filter table to device memory

  auto execpol        = rmm::exec_policy(stream);
  auto strings_column = column_device_view::create(strings.parent(), stream);
  auto d_strings      = *strings_column;

  // create null mask
  rmm::device_buffer null_mask = copy_bitmask(strings.parent(), stream, mr);

  // create offsets column
  filter_fn ffn{d_strings, keep_characters, table.begin(), table.end(), d_replacement};
  auto offsets_transformer_itr =
    thrust::make_transform_iterator(thrust::make_counting_iterator<int32_t>(0), ffn);
  auto offsets_column = make_offsets_child_column(
    offsets_transformer_itr, offsets_transformer_itr + strings_count, mr, stream);
  ffn.d_offsets = offsets_column->view().data<int32_t>();

  // build chars column
  size_type bytes = cudf::detail::get_value<int32_t>(offsets_column->view(), strings_count, stream);
  auto chars_column = strings::detail::create_chars_child_column(
    strings_count, strings.null_count(), bytes, mr, stream);
  ffn.d_chars = chars_column->mutable_view().data<char>();
  thrust::for_each_n(rmm::exec_policy(stream)->on(stream),
                     thrust::make_counting_iterator<cudf::size_type>(0),
                     strings_count,
                     ffn);

  return make_strings_column(strings_count,
                             std::move(offsets_column),
                             std::move(chars_column),
                             strings.null_count(),
                             std::move(null_mask),
                             stream,
                             mr);
}

}  // namespace detail

/**
 * @copydoc cudf::strings::filter_characters
 */
std::unique_ptr<column> filter_characters(
  strings_column_view const& strings,
  std::vector<std::pair<cudf::char_utf8, cudf::char_utf8>> characters_to_filter,
  filter_type keep_characters,
  string_scalar const& replacement,
  rmm::mr::device_memory_resource* mr)
{
  CUDF_FUNC_RANGE();
  return detail::filter_characters(
    strings, characters_to_filter, keep_characters, replacement, 0, mr);
}

}  // namespace strings
}  // namespace cudf
