/*
 * Copyright (c) 2019-2021, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <cudf/column/column_device_view.cuh>
#include <cudf/column/column_factories.hpp>
#include <cudf/detail/null_mask.hpp>
#include <cudf/detail/nvtx/ranges.hpp>
#include <cudf/scalar/scalar_device_view.cuh>
#include <cudf/strings/combine.hpp>
#include <cudf/strings/detail/combine.hpp>
#include <cudf/strings/detail/utilities.cuh>
#include <cudf/strings/string_view.cuh>
#include <cudf/strings/strings_column_view.hpp>
#include <cudf/utilities/error.hpp>

#include <rmm/cuda_stream_view.hpp>
#include <rmm/device_uvector.hpp>
#include <rmm/exec_policy.hpp>

#include <thrust/transform_scan.h>

namespace cudf {
namespace strings {
namespace detail {

std::unique_ptr<column> join_strings(strings_column_view const& strings,
                                     string_scalar const& separator,
                                     string_scalar const& narep,
                                     rmm::cuda_stream_view stream,
                                     rmm::mr::device_memory_resource* mr)
{
  auto strings_count = strings.size();
  if (strings_count == 0) return make_empty_column(data_type{type_id::STRING});

  CUDF_EXPECTS(separator.is_valid(), "Parameter separator must be a valid string_scalar");

  string_view d_separator(separator.data(), separator.size());
  auto d_narep = get_scalar_device_view(const_cast<string_scalar&>(narep));

  auto strings_column = column_device_view::create(strings.parent(), stream);
  auto d_strings      = *strings_column;

  // create an offsets array for building the output memory layout
  rmm::device_uvector<size_type> output_offsets(strings_count + 1, stream);
  auto d_output_offsets = output_offsets.data();
  // using inclusive-scan to compute last entry which is the total size
  thrust::transform_inclusive_scan(
    rmm::exec_policy(stream),
    thrust::make_counting_iterator<size_type>(0),
    thrust::make_counting_iterator<size_type>(strings_count),
    d_output_offsets + 1,
    [d_strings, d_separator, d_narep] __device__(size_type idx) {
      size_type bytes = 0;
      if (d_strings.is_null(idx)) {
        if (!d_narep.is_valid()) return 0;  // skip nulls
        bytes += d_narep.size();
      } else
        bytes += d_strings.element<string_view>(idx).size_bytes();
      if ((idx + 1) < d_strings.size()) bytes += d_separator.size_bytes();
      return bytes;
    },
    thrust::plus<size_type>());

  output_offsets.set_element_to_zero_async(0, stream);
  // total size is the last entry
  size_type const bytes = output_offsets.back_element(stream);

  // build offsets column (only 1 string so 2 offset entries)
  auto offsets_column =
    make_numeric_column(data_type{type_id::INT32}, 2, mask_state::UNALLOCATED, stream, mr);
  auto offsets_view = offsets_column->mutable_view();
  // set the first entry to 0 and the last entry to bytes
  int32_t new_offsets[] = {0, static_cast<int32_t>(bytes)};
  CUDA_TRY(hipMemcpyAsync(offsets_view.data<int32_t>(),
                           new_offsets,
                           sizeof(new_offsets),
                           hipMemcpyHostToDevice,
                           stream.value()));

  // build null mask
  // only one entry so it is either all valid or all null
  auto const null_count =
    static_cast<size_type>(strings.null_count() == strings_count && !narep.is_valid());
  auto null_mask    = null_count
                        ? cudf::detail::create_null_mask(1, cudf::mask_state::ALL_NULL, stream, mr)
                        : rmm::device_buffer{0, stream, mr};
  auto chars_column = create_chars_child_column(bytes, stream, mr);
  auto d_chars      = chars_column->mutable_view().data<char>();
  thrust::for_each_n(
    rmm::exec_policy(stream),
    thrust::make_counting_iterator<size_type>(0),
    strings_count,
    [d_strings, d_separator, d_narep, d_output_offsets, d_chars] __device__(size_type idx) {
      size_type offset = d_output_offsets[idx];
      char* d_buffer   = d_chars + offset;
      if (d_strings.is_null(idx)) {
        if (!d_narep.is_valid())
          return;  // do not write to buffer if element is null (including separator)
        d_buffer = detail::copy_string(d_buffer, d_narep.value());
      } else {
        string_view d_str = d_strings.element<string_view>(idx);
        d_buffer          = detail::copy_string(d_buffer, d_str);
      }
      if ((idx + 1) < d_strings.size()) d_buffer = detail::copy_string(d_buffer, d_separator);
    });

  return make_strings_column(1,
                             std::move(offsets_column),
                             std::move(chars_column),
                             null_count,
                             std::move(null_mask),
                             stream,
                             mr);
}

}  // namespace detail

// external API

std::unique_ptr<column> join_strings(strings_column_view const& strings,
                                     string_scalar const& separator,
                                     string_scalar const& narep,
                                     rmm::mr::device_memory_resource* mr)
{
  CUDF_FUNC_RANGE();
  return detail::join_strings(strings, separator, narep, rmm::cuda_stream_default, mr);
}

}  // namespace strings
}  // namespace cudf
