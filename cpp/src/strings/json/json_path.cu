#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2021, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <cudf/column/column_device_view.cuh>
#include <cudf/column/column_factories.hpp>
#include <cudf/detail/get_value.cuh>
#include <cudf/detail/null_mask.hpp>
#include <cudf/detail/utilities/cuda.cuh>
#include <cudf/detail/utilities/vector_factories.hpp>
#include <cudf/scalar/scalar.hpp>
#include <cudf/strings/string_view.cuh>
#include <cudf/strings/strings_column_view.hpp>
#include <cudf/strings/detail/json.hpp>
#include <cudf/types.hpp>
#include <cudf/utilities/bit.hpp>
#include <cudf/utilities/error.hpp>

#include <io/utilities/parsing_utils.cuh>

#include <rmm/device_uvector.hpp>
#include <rmm/exec_policy.hpp>

#include <thrust/optional.h>

namespace cudf {
namespace strings {
namespace detail {

// debug accessibility

// change to "\n" and 1 to make output more readable
#define DEBUG_NEWLINE
constexpr int DEBUG_NEWLINE_LEN = 0;

enum class path_operator_type { ROOT, CHILD, CHILD_WILDCARD, CHILD_INDEX, ERROR, END };

/**
 * @brief A "command" operator used to query a json string.  A full query is
 * an array of these operators applied to the incoming json string,
 */
struct path_operator {
  CUDA_HOST_DEVICE_CALLABLE path_operator()
    : type(path_operator_type::ERROR), index(-1), expected_type{NONE}
  {
  }
  CUDA_HOST_DEVICE_CALLABLE path_operator(path_operator_type _type,
                                          json_element_type _expected_type = NONE)
    : type(_type), index(-1), expected_type{_expected_type}
  {
  }

  path_operator_type type;  // operator type
  // the expected element type we're applying this operation to.
  // for example:
  //    - you cannot retrieve a subscripted field (eg [5]) from an object.
  //    - you cannot retrieve a field by name (eg  .book) from an array.
  //    - you -can- use .* for both arrays and objects
  // a value of NONE imples any type accepted
  json_element_type expected_type;  // the expected type of the element we're working with
  string_view name;                 // name to match against (if applicable)
  int index;                        // index for subscript operator
};

/**
 * @brief Parsing class that holds the current state of the JSONPath string to be parsed
 * and provides functions for navigating through it. This is only called on the host
 * during the preprocess step which builds a command buffer that the gpu uses.
 */
class path_state : private parser {
 public:
  path_state(const char* _path, size_t _path_len) : parser(_path, _path_len) {}

  // get the next operator in the JSONPath string
  path_operator get_next_operator()
  {
    if (eof()) { return {path_operator_type::END}; }

    switch (*pos++) {
      case '$': return {path_operator_type::ROOT};

      case '.': {
        path_operator op;
        string_view term{".[", 2};
        if (parse_path_name(op.name, term)) {
          // this is another potential use case for __SPARK_BEHAVIORS / configurability
          // Spark currently only handles the wildcard operator inside [*], it does
          // not handle .*
          if (op.name.size_bytes() == 1 && op.name.data()[0] == '*') {
            op.type          = path_operator_type::CHILD_WILDCARD;
            op.expected_type = NONE;
          } else {
            op.type          = path_operator_type::CHILD;
            op.expected_type = OBJECT;
          }
          return op;
        }
      } break;

      // 3 ways this can be used
      // indices:   [0]
      // name:      ['book']
      // wildcard:  [*]
      case '[': {
        path_operator op;
        string_view term{"]", 1};
        bool const is_string = *pos == '\'' ? true : false;
        if (parse_path_name(op.name, term)) {
          pos++;
          if (op.name.size_bytes() == 1 && op.name.data()[0] == '*') {
            op.type          = path_operator_type::CHILD_WILDCARD;
            op.expected_type = NONE;
          } else {
            if (is_string) {
              op.type          = path_operator_type::CHILD;
              op.expected_type = OBJECT;
            } else {
              op.type  = path_operator_type::CHILD_INDEX;
              op.index = cudf::io::parse_numeric<int>(
                op.name.data(), op.name.data() + op.name.size_bytes(), json_opts, -1);
              CUDF_EXPECTS(op.index >= 0, "Invalid numeric index specified in JSONPath");
              op.expected_type = ARRAY;
            }
          }
          return op;
        }
      } break;

      // wildcard operator
      case '*': {
        pos++;
        return path_operator{path_operator_type::CHILD_WILDCARD};
      } break;

      default: CUDF_FAIL("Unrecognized JSONPath operator"); break;
    }
    return {path_operator_type::ERROR};
  }

 private:
  cudf::io::parse_options_view json_opts{',', '\n', '\"', '.'};

  bool parse_path_name(string_view& name, string_view const& terminators)
  {
    switch (*pos) {
      case '*':
        name = string_view(pos, 1);
        pos++;
        break;

      case '\'':
        if (parse_string(name, false, '\'') != parse_result::SUCCESS) { return false; }
        break;

      default: {
        size_t const chars_left = input_len - (pos - input);
        char const* end         = std::find_first_of(
          pos, pos + chars_left, terminators.data(), terminators.data() + terminators.size_bytes());
        if (end) {
          name = string_view(pos, end - pos);
          pos  = end;
        } else {
          name = string_view(pos, chars_left);
          pos  = input + input_len;
        }
        break;
      }
    }

    // an empty name is not valid
    CUDF_EXPECTS(name.size_bytes() > 0, "Invalid empty name in JSONPath query string");

    return true;
  }
};

/**
 * @brief Preprocess the incoming JSONPath string on the host to generate a
 * command buffer for use by the GPU.
 *
 * @param json_path The incoming json path
 * @param stream Cuda stream to perform any gpu actions on
 * @returns A pair containing the command buffer, and maximum stack depth required.
 */
std::pair<thrust::optional<rmm::device_uvector<path_operator>>, int> build_command_buffer(
  cudf::string_scalar const& json_path, rmm::cuda_stream_view stream)
{
  std::string h_json_path = json_path.to_string(stream);
  path_state p_state(h_json_path.data(), static_cast<size_type>(h_json_path.size()));

  std::vector<path_operator> h_operators;

  path_operator op;
  int max_stack_depth = 1;
  do {
    op = p_state.get_next_operator();
    if (op.type == path_operator_type::ERROR) {
      CUDF_FAIL("Encountered invalid JSONPath input string");
    }
    if (op.type == path_operator_type::CHILD_WILDCARD) { max_stack_depth++; }
    // convert pointer to device pointer
    if (op.name.size_bytes() > 0) {
      op.name =
        string_view(json_path.data() + (op.name.data() - h_json_path.data()), op.name.size_bytes());
    }
    if (op.type == path_operator_type::ROOT) {
      CUDF_EXPECTS(h_operators.size() == 0, "Root operator ($) can only exist at the root");
    }
    // if we havent' gotten a root operator to start, and we're not empty, quietly push a
    // root operator now.
    if (h_operators.size() == 0 && op.type != path_operator_type::ROOT &&
        op.type != path_operator_type::END) {
      h_operators.push_back(path_operator{path_operator_type::ROOT});
    }
    h_operators.push_back(op);
  } while (op.type != path_operator_type::END);

  auto const is_empty = h_operators.size() == 1 && h_operators[0].type == path_operator_type::END;
  return is_empty
           ? std::make_pair(thrust::nullopt, 0)
           : std::make_pair(
               thrust::make_optional(cudf::detail::make_device_uvector_sync(h_operators, stream)),
               max_stack_depth);
}

/**
 * @brief Parse a single json string using the provided command buffer
 *
 * @param j_state The incoming json string and associated parser
 * @param commands The command buffer to be applied to the string. Always ends with a
 * path_operator_type::END
 * @param output Buffer user to store the results of the query
 * @returns A result code indicating success/fail/empty.
 */
template <int max_command_stack_depth>
__device__ parse_result parse_json_path(json_state& j_state,
                                        path_operator const* commands,
                                        json_output& output)
{
  // manually maintained context stack in lieu of calling parse_json_path recursively.
  struct context {
    json_state j_state;
    path_operator const* commands;
    bool list_element;
    bool state_flag;
  };
  context stack[max_command_stack_depth];
  int stack_pos     = 0;
  auto push_context = [&stack, &stack_pos](json_state const& _j_state,
                                           path_operator const* _commands,
                                           bool _list_element = false,
                                           bool _state_flag   = false) {
    if (stack_pos == max_command_stack_depth - 1) { return false; }
    stack[stack_pos++] = context{_j_state, _commands, _list_element, _state_flag};
    return true;
  };
  auto pop_context = [&stack, &stack_pos](context& c) {
    if (stack_pos > 0) {
      c = stack[--stack_pos];
      return true;
    }
    return false;
  };
  push_context(j_state, commands, false);

  parse_result last_result = parse_result::SUCCESS;
  context ctx;
  int element_count = 0;
  while (pop_context(ctx)) {
    path_operator op = *ctx.commands;

    switch (op.type) {
      // whatever the first object is
      case path_operator_type::ROOT:
        PARSE_TRY(ctx.j_state.next_element());
        push_context(ctx.j_state, ctx.commands + 1);
        break;

      // .name
      // ['name']
      // [1]
      // will return a single thing
      case path_operator_type::CHILD: {
        PARSE_TRY(ctx.j_state.child_element(op.expected_type));
        if (last_result == parse_result::SUCCESS) {
          PARSE_TRY(ctx.j_state.next_matching_element(op.name, true));
          if (last_result == parse_result::SUCCESS) {
            push_context(ctx.j_state, ctx.commands + 1, ctx.list_element);
          }
        }
      } break;

      // .*
      // [*]
      // will return an array of things
      case path_operator_type::CHILD_WILDCARD: {
        // if we're on the first element of this wildcard
        if (!ctx.state_flag) {
          // we will only ever be returning 1 array
          if (!ctx.list_element) { output.add_output({"[" DEBUG_NEWLINE, 1 + DEBUG_NEWLINE_LEN}); }

          // step into the child element
          PARSE_TRY(ctx.j_state.child_element(op.expected_type));
          if (last_result == parse_result::EMPTY) {
            if (!ctx.list_element) {
              output.add_output({"]" DEBUG_NEWLINE, 1 + DEBUG_NEWLINE_LEN});
            }
            last_result = parse_result::SUCCESS;
            break;
          }

          // first element
          PARSE_TRY(ctx.j_state.next_matching_element({"*", 1}, true));
          if (last_result == parse_result::EMPTY) {
            if (!ctx.list_element) {
              output.add_output({"]" DEBUG_NEWLINE, 1 + DEBUG_NEWLINE_LEN});
            }
            last_result = parse_result::SUCCESS;
            break;
          }

          // re-push ourselves
          push_context(ctx.j_state, ctx.commands, ctx.list_element, true);
          // push the next command
          push_context(ctx.j_state, ctx.commands + 1, true);
        } else {
          // next element
          PARSE_TRY(ctx.j_state.next_matching_element({"*", 1}, false));
          if (last_result == parse_result::EMPTY) {
            if (!ctx.list_element) {
              output.add_output({"]" DEBUG_NEWLINE, 1 + DEBUG_NEWLINE_LEN});
            }
            last_result = parse_result::SUCCESS;
            break;
          }

          // re-push ourselves
          push_context(ctx.j_state, ctx.commands, ctx.list_element, true);
          // push the next command
          push_context(ctx.j_state, ctx.commands + 1, true);
        }
      } break;

      // [0]
      // [1]
      // etc
      // returns a single thing
      case path_operator_type::CHILD_INDEX: {
        PARSE_TRY(ctx.j_state.child_element(op.expected_type));
        if (last_result == parse_result::SUCCESS) {
          string_view const any{"*", 1};
          PARSE_TRY(ctx.j_state.next_matching_element(any, true));
          if (last_result == parse_result::SUCCESS) {
            int idx;
            for (idx = 1; idx <= op.index; idx++) {
              PARSE_TRY(ctx.j_state.next_matching_element(any, false));
              if (last_result == parse_result::EMPTY) { break; }
            }
            // if we didn't end up at the index we requested, this is an invalid index
            if (idx - 1 != op.index) { return parse_result::ERROR; }
            push_context(ctx.j_state, ctx.commands + 1, ctx.list_element);
          }
        }
      } break;

      // some sort of error.
      case path_operator_type::ERROR: return parse_result::ERROR; break;

      // END case
      default: {
        if (ctx.list_element && element_count > 0) {
          output.add_output({"," DEBUG_NEWLINE, 1 + DEBUG_NEWLINE_LEN});
        }
        PARSE_TRY(ctx.j_state.extract_element(&output, ctx.list_element));
        if (ctx.list_element && last_result != parse_result::EMPTY) { element_count++; }
      } break;
    }
  }

  return parse_result::SUCCESS;
}

// hardcoding this for now. to reach a stack depth of 8 would require
// a JSONPath containing 7 nested wildcards so this is probably reasonable.
constexpr int max_command_stack_depth = 8;

/**
 * @brief Parse a single json string using the provided command buffer
 *
 * This function exists primarily as a shim for debugging purposes.
 *
 * @param input The incoming json string
 * @param input_len Size of the incoming json string
 * @param commands The command buffer to be applied to the string. Always ends with a
 * path_operator_type::END
 * @param out_buf Buffer user to store the results of the query (nullptr in the size computation
 * step)
 * @param out_buf_size Size of the output buffer
 * @returns A pair containing the result code the output buffer.
 */
__device__ thrust::pair<parse_result, json_output> get_json_object_single(
  char const* input,
  size_t input_len,
  path_operator const* const commands,
  char* out_buf,
  size_t out_buf_size)
{
  json_state j_state(input, input_len);
  json_output output{out_buf_size, out_buf};

  auto const result = parse_json_path<max_command_stack_depth>(j_state, commands, output);

  return {result, output};
}

/**
 * @brief Kernel for running the JSONPath query.
 *
 * This kernel operates in a 2-pass way.  On the first pass, it computes
 * output sizes.  On the second pass it fills in the provided output buffers
 * (chars and validity)
 *
 * @param col Device view of the incoming string
 * @param commands JSONPath command buffer
 * @param output_offsets Buffer used to store the string offsets for the results of the query
 * @param out_buf Buffer used to store the results of the query
 * @param out_validity Output validity buffer
 * @param out_valid_count Output count of # of valid bits
 */
template <int block_size>
__launch_bounds__(block_size) __global__
  void get_json_object_kernel(column_device_view col,
                              path_operator const* const commands,
                              offset_type* output_offsets,
                              thrust::optional<char*> out_buf,
                              thrust::optional<bitmask_type*> out_validity,
                              thrust::optional<size_type*> out_valid_count)
{
  size_type tid    = threadIdx.x + (blockDim.x * blockIdx.x);
  size_type stride = blockDim.x * gridDim.x;

  if (out_valid_count.has_value()) { *(out_valid_count.value()) = 0; }
  size_type warp_valid_count{0};

  auto active_threads = __ballot_sync(0xffffffff, tid < col.size());
  while (tid < col.size()) {
    bool is_valid         = false;
    string_view const str = col.element<string_view>(tid);
    size_type output_size = 0;
    if (str.size_bytes() > 0) {
      char* dst = out_buf.has_value() ? out_buf.value() + output_offsets[tid] : nullptr;
      size_t const dst_size =
        out_buf.has_value() ? output_offsets[tid + 1] - output_offsets[tid] : 0;

      parse_result result;
      json_output out;
      thrust::tie(result, out) =
        get_json_object_single(str.data(), str.size_bytes(), commands, dst, dst_size);
      output_size = out.output_len.value_or(0);
      if (out.output_len.has_value() && result == parse_result::SUCCESS) { is_valid = true; }
    }

    // filled in only during the precompute step. during the compute step, the offsets
    // are fed back in so we do -not- want to write them out
    if (!out_buf.has_value()) { output_offsets[tid] = static_cast<offset_type>(output_size); }

    // validity filled in only during the output step
    if (out_validity.has_value()) {
      uint32_t mask = __ballot_sync(active_threads, is_valid);
      // 0th lane of the warp writes the validity
      if (!(tid % cudf::detail::warp_size)) {
        out_validity.value()[cudf::word_index(tid)] = mask;
        warp_valid_count += __popc(mask);
      }
    }

    tid += stride;
    active_threads = __ballot_sync(active_threads, tid < col.size());
  }

  // sum the valid counts across the whole block
  if (out_valid_count) {
    size_type block_valid_count =
      cudf::detail::single_lane_block_sum_reduce<block_size, 0>(warp_valid_count);
    if (threadIdx.x == 0) { atomicAdd(out_valid_count.value(), block_valid_count); }
  }
}

/**
 * @copydoc cudf::strings::detail::get_json_object
 */
std::unique_ptr<cudf::column> get_json_object(cudf::strings_column_view const& col,
                                              cudf::string_scalar const& json_path,
                                              rmm::cuda_stream_view stream,
                                              rmm::mr::device_memory_resource* mr)
{
  // preprocess the json_path into a command buffer
  auto preprocess = build_command_buffer(json_path, stream);
  CUDF_EXPECTS(std::get<1>(preprocess) <= max_command_stack_depth,
               "Encountered JSONPath string that is too complex");

  // allocate output offsets buffer.
  auto offsets = cudf::make_fixed_width_column(
    data_type{type_id::INT32}, col.size() + 1, mask_state::UNALLOCATED, stream, mr);
  cudf::mutable_column_view offsets_view(*offsets);

  // if the query is empty, return a string column containing all nulls
  if (!std::get<0>(preprocess).has_value()) {
    return std::make_unique<column>(
      data_type{type_id::STRING},
      col.size(),
      rmm::device_buffer{0, stream, mr},  // no data
      cudf::detail::create_null_mask(col.size(), mask_state::ALL_NULL, stream, mr),
      col.size());  // null count
  }

  constexpr int block_size = 512;
  cudf::detail::grid_1d const grid{col.size(), block_size};

  auto cdv = column_device_view::create(col.parent(), stream);

  // preprocess sizes (returned in the offsets buffer)
  get_json_object_kernel<block_size>
    <<<grid.num_blocks, grid.num_threads_per_block, 0, stream.value()>>>(
      *cdv,
      std::get<0>(preprocess).value().data(),
      offsets_view.head<offset_type>(),
      thrust::nullopt,
      thrust::nullopt,
      thrust::nullopt);

  // convert sizes to offsets
  thrust::exclusive_scan(rmm::exec_policy(stream),
                         offsets_view.head<offset_type>(),
                         offsets_view.head<offset_type>() + col.size() + 1,
                         offsets_view.head<offset_type>(),
                         0);
  size_type const output_size =
    cudf::detail::get_value<offset_type>(offsets_view, col.size(), stream);

  // allocate output string column
  auto chars = cudf::make_fixed_width_column(
    data_type{type_id::INT8}, output_size, mask_state::UNALLOCATED, stream, mr);

  // potential optimization : if we know that all outputs are valid, we could skip creating
  // the validity mask altogether
  rmm::device_buffer validity =
    cudf::detail::create_null_mask(col.size(), mask_state::UNINITIALIZED, stream, mr);

  // compute results
  cudf::mutable_column_view chars_view(*chars);
  rmm::device_scalar<size_type> d_valid_count{0, stream};
  get_json_object_kernel<block_size>
    <<<grid.num_blocks, grid.num_threads_per_block, 0, stream.value()>>>(
      *cdv,
      std::get<0>(preprocess).value().data(),
      offsets_view.head<offset_type>(),
      chars_view.head<char>(),
      static_cast<bitmask_type*>(validity.data()),
      d_valid_count.data());

  return make_strings_column(col.size(),
                             std::move(offsets),
                             std::move(chars),
                             col.size() - d_valid_count.value(),
                             std::move(validity),
                             stream,
                             mr);
}

}  // namespace detail

/**
 * @copydoc cudf::strings::get_json_object
 */
std::unique_ptr<cudf::column> get_json_object(cudf::strings_column_view const& col,
                                              cudf::string_scalar const& json_path,
                                              rmm::mr::device_memory_resource* mr)
{
  CUDF_FUNC_RANGE();
  return detail::get_json_object(col, json_path, 0, mr);
}

}  // namespace strings
}  // namespace cudf

