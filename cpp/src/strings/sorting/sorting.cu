/*
 * Copyright (c) 2019, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <cudf/column/column_device_view.cuh>
#include <cudf/detail/gather.hpp>
#include <cudf/strings/sorting.hpp>
#include <cudf/strings/string_view.cuh>
#include <cudf/strings/strings_column_view.hpp>

#include <rmm/thrust_rmm_allocator.h>
#include <rmm/cuda_stream_view.hpp>

#include <thrust/sequence.h>
#include <thrust/sort.h>

namespace cudf {
namespace strings {
namespace detail {
// return sorted version of the given strings column
std::unique_ptr<cudf::column> sort(strings_column_view strings,
                                   sort_type stype,
                                   cudf::order order,
                                   cudf::null_order null_order,
                                   rmm::cuda_stream_view stream,
                                   rmm::mr::device_memory_resource* mr)
{
  auto execpol        = rmm::exec_policy(stream);
  auto strings_column = column_device_view::create(strings.parent(), stream);
  auto d_column       = *strings_column;

  // sort the indices of the strings
  size_type num_strings = strings.size();
  rmm::device_vector<size_type> indices(num_strings);
  thrust::sequence(execpol->on(stream.value()), indices.begin(), indices.end());
  thrust::sort(execpol->on(stream.value()),
               indices.begin(),
               indices.end(),
               [d_column, stype, order, null_order] __device__(size_type lhs, size_type rhs) {
                 bool lhs_null{d_column.is_null(lhs)};
                 bool rhs_null{d_column.is_null(rhs)};
                 if (lhs_null || rhs_null)
                   return (null_order == cudf::null_order::BEFORE ? !rhs_null : !lhs_null);
                 string_view lhs_str = d_column.element<string_view>(lhs);
                 string_view rhs_str = d_column.element<string_view>(rhs);
                 int cmp             = 0;
                 if (stype & sort_type::length) cmp = lhs_str.length() - rhs_str.length();
                 if (stype & sort_type::name) cmp = lhs_str.compare(rhs_str);
                 return (order == cudf::order::ASCENDING ? (cmp < 0) : (cmp > 0));
               });

  // create a column_view as a wrapper of these indices
  column_view indices_view(
    data_type{type_id::INT32}, num_strings, indices.data().get(), nullptr, 0);
  // now build a new strings column from the indices
  auto table_sorted = cudf::detail::gather(table_view{{strings.parent()}},
                                           indices_view,
                                           cudf::detail::out_of_bounds_policy::NULLIFY,
                                           cudf::detail::negative_index_policy::NOT_ALLOWED,
                                           stream,
                                           mr)
                        ->release();
  return std::move(table_sorted.front());
}

}  // namespace detail
}  // namespace strings
}  // namespace cudf
