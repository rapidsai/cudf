/*
 * Copyright (c) 2020, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <cudf/column/column.hpp>
#include <cudf/column/column_factories.hpp>
#include <cudf/column/column_device_view.cuh>
#include <cudf/strings/strings_column_view.hpp>
#include <cudf/strings/string_view.cuh>
#include <cudf/strings/case.hpp>
#include <cudf/utilities/error.hpp>
#include <strings/char_types/is_flags.h>
#include <strings/utilities.hpp>
#include <strings/utilities.cuh>


namespace cudf
{
namespace strings
{
namespace detail
{
namespace { // anonym.

  //base class for probing string
  //manipulation memory load requirements;
  //and for executing string modification:
  //
  struct probe_execute_base
  {
    using char_info = thrust::pair<uint32_t,detail::character_flags_table_type>;
    
    probe_execute_base(column_device_view const d_column,
                       int32_t const* d_offsets = nullptr,
                       char* d_chars = nullptr):
      d_column_(d_column),
      d_flags_(get_character_flags_table()),      // set flag table
      d_case_table_(get_character_cases_table()), // set case table
      d_offsets_(d_offsets),
      d_chars_(d_chars)
    {
    }

    __host__ __device__
    column_device_view const get_column(void) const
    {
      return d_column_;
    }

    __device__
    char_info get_char_info(char_utf8 chr) const
    {
      uint32_t code_point = detail::utf8_to_codepoint(chr);
      detail::character_flags_table_type flag = code_point <= 0x00FFFF ? d_flags_[code_point] : 0;
      return char_info{code_point,flag};
    }

    __device__
    char_utf8 convert_char(char_info const& info) const
    {
      return detail::codepoint_to_utf8(d_case_table_[info.first]);
    }

    __device__ char* get_output_ptr(size_type idx)
    {
      return d_chars_ && d_offsets_ ? d_chars_ + d_offsets_[idx] : nullptr;
    }
  private:
    column_device_view const d_column_;
    character_flags_table_type const* d_flags_;
    character_cases_table_type const* d_case_table_;
    int32_t const* d_offsets_;
    char* d_chars_;
  };


  //class that factors out the common inside-loop behavior
  //of operator() between capitalize's `probe` and `execute`;
  //(public inheritance to allow getters pass-through
  //in derived classes);
  //
  struct probe_execute_capitalize: public probe_execute_base
  {
    explicit probe_execute_capitalize(column_device_view const d_column):
      probe_execute_base(d_column)
    {
    }

    probe_execute_capitalize(column_device_view const d_column,
                             int32_t const* d_offsets,
                             char* d_chars):
      probe_execute_base(d_column, d_offsets, d_chars)
    {
    }
    
    __device__
    char_utf8 generate_chr(string_view::const_iterator itr, string_view d_str) const {
      auto the_chr = *itr;

      auto pair_char_info = get_char_info(the_chr);
      detail::character_flags_table_type flag = pair_char_info.second;
         
      if( (itr == d_str.begin()) ? IS_LOWER(flag) : IS_UPPER(flag) )
        the_chr = convert_char(pair_char_info);

      return the_chr;
    }
  };


  //functor for probing string capitalization
  //requirements:
  //(private inheritance to prevent polymorphic use,
  // a requirement that came up in code review)
  //
  struct probe_capitalize: private probe_execute_capitalize
  {
    explicit probe_capitalize(column_device_view const d_column):
      //probe_execute_base(d_column)
      probe_execute_capitalize(d_column)
    {  
    }

     __device__
     int32_t operator()(size_type idx) const {
       if( get_column().is_null(idx) )
         return 0; // null string
      
       string_view d_str = get_column().template element<string_view>(idx);
       int32_t bytes = 0;
      
       for( auto itr = d_str.begin(); itr != d_str.end(); ++itr ) {         
         bytes += detail::bytes_in_char_utf8(generate_chr(itr, d_str));
       }
       return bytes;
    }
  };

  //functor for executing string capitalization:
  //(private inheritance to prevent polymorphic use,
  // a requirement that came up in code review)
  //
  struct execute_capitalize: private probe_execute_capitalize
  {
    execute_capitalize(column_device_view const d_column,
                       int32_t const* d_offsets,
                       char* d_chars):
      //probe_execute_base(d_column, d_offsets, d_chars)
      probe_execute_capitalize(d_column, d_offsets, d_chars)
    {
    }
    
    __device__
    int32_t operator()(size_type idx) {
      if( get_column().is_null(idx) )
        return 0; // null string
      
      string_view d_str = get_column().template element<string_view>(idx);
      char* d_buffer = get_output_ptr(idx);
      
      for( auto itr = d_str.begin(); itr != d_str.end(); ++itr ) {
        d_buffer += detail::from_char_utf8(generate_chr(itr, d_str), d_buffer);
      }
      return 0;
    }
  };


  //class that factors out the common inside-loop behavior
  //of operator() between title's `probe` and `execute`;
  //(public inheritance to allow getters pass-through
  //in derived classes);
  //
  struct probe_execute_title: public probe_execute_base
  {
    explicit probe_execute_title(column_device_view const d_column):
      probe_execute_base(d_column)
    {
    }

    probe_execute_title(column_device_view const d_column,
                             int32_t const* d_offsets,
                             char* d_chars):
      probe_execute_base(d_column, d_offsets, d_chars)
    {
    }
    
    __device__
    thrust::pair<char_utf8, bool> generate_chr(string_view::const_iterator itr, string_view d_str, bool bcapnext) const {
      auto the_chr = *itr;

      auto pair_char_info = get_char_info(the_chr);
      detail::character_flags_table_type flag = pair_char_info.second;

      if( !IS_ALPHA(flag) ) {
        bcapnext = true;
      } else {
        if( bcapnext ? IS_LOWER(flag) : IS_UPPER(flag) )
          the_chr = convert_char(pair_char_info);
           
        bcapnext = false;
      }

      return thrust::make_pair(the_chr, bcapnext);
    }
  };


  //functor for probing string title-ization
  //requirements:
  //(private inheritance to prevent polymorphic use,
  // a requirement that came up in code review)
  //
  struct probe_title: private probe_execute_title
  {
    explicit probe_title(column_device_view const d_column):
      probe_execute_title(d_column)
    {  
    }
    
     __device__
     int32_t operator()(size_type idx) const {
       if( get_column().is_null(idx) )
         return 0; // null string
      
       string_view d_str = get_column().template element<string_view>(idx);
       int32_t bytes = 0;

       bool bcapnext = true;
       for( auto itr = d_str.begin(); itr != d_str.end(); ++itr ) {
         auto pair_char_flag = generate_chr(itr, d_str, bcapnext);
         bcapnext = pair_char_flag.second;
        
         bytes += detail::bytes_in_char_utf8(pair_char_flag.first);
       }
       return bytes;
    }
  };

  //functor for executing string title-ization:
  //(private inheritance to prevent polymorphic use,
  // a requirement that came up in code review)
  //
  struct execute_title: private probe_execute_title
  {
    execute_title(column_device_view const d_column,
                  int32_t const* d_offsets,
                  char* d_chars):
      probe_execute_title(d_column, d_offsets, d_chars)
    {
    }
    
    __device__
    int32_t operator()(size_type idx) {
      if( get_column().is_null(idx) )
        return 0; // null string
      
      string_view d_str = get_column().template element<string_view>(idx);
      char* d_buffer = get_output_ptr(idx);

      bool bcapnext = true;
      for( auto itr = d_str.begin(); itr != d_str.end(); ++itr ) {
        auto pair_char_flag = generate_chr(itr, d_str, bcapnext);
        bcapnext = pair_char_flag.second;
        
        d_buffer += detail::from_char_utf8(pair_char_flag.first, d_buffer);
      }
      return 0;
    }
  };
       
}//anonym.

template<typename device_probe_functor,
         typename device_execute_functor>
std::unique_ptr<column> modify_strings( strings_column_view const& strings,
                                        rmm::mr::device_memory_resource* mr = rmm::mr::get_default_resource(),
                                        hipStream_t stream = 0)
{
  auto strings_count = strings.size();
  if( strings_count == 0 )
    return detail::make_empty_strings_column(mr,stream);

  auto execpol = rmm::exec_policy(stream);
  
  auto strings_column = column_device_view::create(strings.parent(),stream);
  auto d_column = *strings_column;

  // copy null mask
  rmm::device_buffer null_mask = copy_bitmask(strings.parent(),stream,mr);
  // get the lookup tables used for case conversion  

  device_probe_functor d_probe_fctr{d_column};

  // build offsets column -- calculate the size of each output string
  auto offsets_transformer_itr = thrust::make_transform_iterator( thrust::make_counting_iterator<size_type>(0), d_probe_fctr);
  auto offsets_column = detail::make_offsets_child_column(offsets_transformer_itr,
                                                          offsets_transformer_itr+strings_count,
                                                          mr, stream);
  auto offsets_view = offsets_column->view();
  auto d_new_offsets = offsets_view.template data<int32_t>();//not sure why this requires `.template` and the next one (`d_chars = ...`) doesn't

  // build the chars column -- convert characters based on case_flag parameter
  size_type bytes = thrust::device_pointer_cast(d_new_offsets)[strings_count];
  auto chars_column = strings::detail::create_chars_child_column( strings_count, d_column.null_count(), bytes, mr, stream );
  auto chars_view = chars_column->mutable_view();
  auto d_chars = chars_view.data<char>();

  device_execute_functor d_execute_fctr{d_column,
      d_new_offsets,
      d_chars};
  
  thrust::for_each_n(execpol->on(stream),
                     thrust::make_counting_iterator<size_type>(0), strings_count, d_execute_fctr);
  
  //
  return make_strings_column(strings_count, std::move(offsets_column), std::move(chars_column),
                             d_column.null_count(), std::move(null_mask), stream, mr);
}

}//namespace detail

std::unique_ptr<column> capitalize( strings_column_view const& strings,
                                    rmm::mr::device_memory_resource* mr)
{
  return detail::modify_strings<detail::probe_capitalize, detail::execute_capitalize>(strings, mr);
}

std::unique_ptr<column> title( strings_column_view const& strings,
                               rmm::mr::device_memory_resource* mr)
{
  return detail::modify_strings<detail::probe_title, detail::execute_title>(strings, mr);
}
  
}//namespace strings
}//namespace cudf
