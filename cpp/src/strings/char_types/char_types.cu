/*
 * Copyright (c) 2019, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <cudf/column/column.hpp>
#include <cudf/column/column_factories.hpp>
#include <cudf/column/column_device_view.cuh>
#include <cudf/strings/strings_column_view.hpp>
#include <cudf/strings/string_view.cuh>
#include <cudf/strings/char_types/char_types.hpp>
#include <cudf/wrappers/bool.hpp>
#include "../utilities.hpp"
#include "../utilities.cuh"

//
namespace cudf
{
namespace strings
{
namespace detail
{
//
std::unique_ptr<column> all_characters_of_type( strings_column_view const& strings,
                                                string_character_types types,
                                                rmm::mr::device_memory_resource* mr = rmm::mr::get_default_resource(),
                                                hipStream_t stream = 0)
{
    auto strings_count = strings.size();
    auto strings_column = column_device_view::create(strings.parent(),stream);
    auto d_column = *strings_column;

    // create output column
    auto results = make_numeric_column( data_type{BOOL8}, strings_count,
        copy_bitmask(strings.parent(),stream,mr), strings.null_count(), stream, mr);
    auto results_view = results->mutable_view();
    auto d_results = results_view.data<experimental::bool8>();
    // get the static character types table
    auto d_flags = detail::get_character_flags_table();
    // set the output values by checking the character types for each string
    thrust::transform(rmm::exec_policy(stream)->on(stream),
        thrust::make_counting_iterator<size_type>(0),
        thrust::make_counting_iterator<size_type>(strings_count),
        d_results,
        [d_column, d_flags, types, d_results] __device__(size_type idx){
            if( d_column.is_null(idx) )
                return false;
            auto d_str = d_column.element<string_view>(idx);
            bool check = !d_str.empty(); // positive result requires at least one character
            for( auto itr = d_str.begin(); check && (itr != d_str.end()); ++itr )
            {
                auto code_point = detail::utf8_to_codepoint(*itr);
                // lookup flags in table by code-point
                auto flag = code_point <= 0x00FFFF ? d_flags[code_point] : 0;
                check = (types & flag) > 0;
            }
            return check;
        });
    //
    results->set_null_count(strings.null_count());
    return results;
}

} // namespace detail

// external API

std::unique_ptr<column> all_characters_of_type( strings_column_view const& strings,
                                                string_character_types types,
                                                rmm::mr::device_memory_resource* mr)
{
    return detail::all_characters_of_type(strings, types, mr);
}

} // namespace strings
} // namespace cudf
