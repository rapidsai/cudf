/*
 * Copyright (c) 2019-2020, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <cudf/column/column.hpp>
#include <cudf/column/column_device_view.cuh>
#include <cudf/column/column_factories.hpp>
#include <cudf/detail/null_mask.hpp>
#include <cudf/detail/nvtx/ranges.hpp>
#include <cudf/strings/char_types/char_types.hpp>
#include <cudf/strings/detail/utilities.hpp>
#include <cudf/strings/string.cuh>
#include <cudf/strings/string_view.cuh>
#include <cudf/strings/strings_column_view.hpp>
#include <strings/utilities.cuh>
#include <strings/utilities.hpp>

#include <rmm/cuda_stream_view.hpp>

#include <thrust/logical.h>

namespace cudf {
namespace strings {
namespace detail {
//
std::unique_ptr<column> all_characters_of_type(
  strings_column_view const& strings,
  string_character_types types,
  string_character_types verify_types,
  rmm::mr::device_memory_resource* mr = rmm::mr::get_current_device_resource(),
  hipStream_t stream                 = 0)
{
  auto strings_count  = strings.size();
  auto strings_column = column_device_view::create(strings.parent(), stream);
  auto d_column       = *strings_column;

  // create output column
  auto results = make_numeric_column(
    data_type{type_id::BOOL8},
    strings_count,
    cudf::detail::copy_bitmask(strings.parent(), rmm::cuda_stream_view{stream}, mr),
    strings.null_count(),
    stream,
    mr);
  auto results_view = results->mutable_view();
  auto d_results    = results_view.data<bool>();
  // get the static character types table
  auto d_flags = detail::get_character_flags_table();
  // set the output values by checking the character types for each string
  thrust::transform(rmm::exec_policy(stream)->on(stream),
                    thrust::make_counting_iterator<size_type>(0),
                    thrust::make_counting_iterator<size_type>(strings_count),
                    d_results,
                    [d_column, d_flags, types, verify_types, d_results] __device__(size_type idx) {
                      if (d_column.is_null(idx)) return false;
                      auto d_str            = d_column.element<string_view>(idx);
                      bool check            = !d_str.empty();  // require at least one character
                      size_type check_count = 0;
                      for (auto itr = d_str.begin(); check && (itr != d_str.end()); ++itr) {
                        auto code_point = detail::utf8_to_codepoint(*itr);
                        // lookup flags in table by code-point
                        auto flag = code_point <= 0x00FFFF ? d_flags[code_point] : 0;
                        if ((verify_types & flag) ||                   // should flag be verified
                            (flag == 0 && verify_types == ALL_TYPES))  // special edge case
                        {
                          check = (types & flag) > 0;
                          ++check_count;
                        }
                      }
                      return check && (check_count > 0);
                    });
  //
  results->set_null_count(strings.null_count());
  return results;
}

namespace {

/**
 * @brief Removes individual characters from a strings column based on character type.
 *
 * Types to remove are specified by `types_to_remove` OR
 * types to not remove are specified by `types_to_keep`.
 *
 * This is called twice. The first pass calculates the size of each output string.
 * The final pass copies the results to the output strings column memory.
 */
struct filter_chars_fn {
  column_device_view const d_column;
  character_flags_table_type const* d_flags;
  string_character_types const types_to_remove;
  string_character_types const types_to_keep;
  string_view const d_replacement;  ///< optional replacement for removed characters
  int32_t* d_offsets{};             ///< size of the output string stored here during first pass
  char* d_chars{};                  ///< this is null only during the first pass

  /**
   * @brief Returns true if the given character should be replaced.
   */
  __device__ bool replace_char(char_utf8 ch)
  {
    auto const code_point = detail::utf8_to_codepoint(ch);
    auto const flag       = code_point <= 0x00FFFF ? d_flags[code_point] : 0;
    if (flag == 0)  // all types pass unless specifically identified
      return (types_to_remove == ALL_TYPES);
    if (types_to_keep == ALL_TYPES)  // filter case
      return (types_to_remove & flag) != 0;
    return (types_to_keep & flag) == 0;  // keep case
  }

  __device__ void operator()(size_type idx)
  {
    if (d_column.is_null(idx)) {
      if (!d_chars) d_offsets[idx] = 0;
      return;
    }
    auto const d_str  = d_column.element<string_view>(idx);
    auto const in_ptr = d_str.data();
    auto out_ptr      = d_chars ? d_chars + d_offsets[idx] : nullptr;
    auto nbytes       = d_str.size_bytes();

    for (auto itr = d_str.begin(); itr != d_str.end(); ++itr) {
      auto const char_size = bytes_in_char_utf8(*itr);
      string_view const d_newchar =
        replace_char(*itr) ? d_replacement : string_view(in_ptr + itr.byte_offset(), char_size);
      nbytes += d_newchar.size_bytes() - char_size;
      if (out_ptr) out_ptr = cudf::strings::detail::copy_string(out_ptr, d_newchar);
    }
    if (!out_ptr) d_offsets[idx] = nbytes;
  }
};

}  // namespace

std::unique_ptr<column> filter_characters_of_type(strings_column_view const& strings,
                                                  string_character_types types_to_remove,
                                                  string_scalar const& replacement,
                                                  string_character_types types_to_keep,
                                                  hipStream_t stream,
                                                  rmm::mr::device_memory_resource* mr)
{
  CUDF_EXPECTS(replacement.is_valid(), "Parameter replacement must be valid");
  if (types_to_remove == ALL_TYPES)
    CUDF_EXPECTS(types_to_keep != ALL_TYPES,
                 "Parameters types_to_remove and types_to_keep must not be both ALL_TYPES");
  else
    CUDF_EXPECTS(types_to_keep == ALL_TYPES,
                 "One of parameter types_to_remove and types_to_keep must be set to ALL_TYPES");

  auto const strings_count = strings.size();
  if (strings_count == 0) return make_empty_column(cudf::data_type{cudf::type_id::STRING});

  auto strings_column = cudf::column_device_view::create(strings.parent(), stream);
  cudf::string_view d_replacement(replacement.data(), replacement.size());
  filter_chars_fn filterer{*strings_column,
                           detail::get_character_flags_table(),
                           types_to_remove,
                           types_to_keep,
                           d_replacement};

  // copy null mask from input column
  rmm::device_buffer null_mask =
    cudf::detail::copy_bitmask(strings.parent(), rmm::cuda_stream_view{stream}, mr);

  // this utility calls filterer to build the offsets and chars columns
  auto children = cudf::strings::detail::make_strings_children(
    filterer, strings_count, strings.null_count(), mr, stream);

  // return new strings column
  return make_strings_column(strings_count,
                             std::move(children.first),
                             std::move(children.second),
                             strings.null_count(),
                             std::move(null_mask),
                             stream,
                             mr);
}

std::unique_ptr<column> is_integer(
  strings_column_view const& strings,
  rmm::mr::device_memory_resource* mr = rmm::mr::get_current_device_resource(),
  hipStream_t stream                 = 0)
{
  auto strings_column = column_device_view::create(strings.parent(), stream);
  auto d_column       = *strings_column;
  // create output column
  auto results = make_numeric_column(
    data_type{type_id::BOOL8},
    strings.size(),
    cudf::detail::copy_bitmask(strings.parent(), rmm::cuda_stream_view{stream}, mr),
    strings.null_count(),
    stream,
    mr);
  auto d_results = results->mutable_view().data<bool>();
  thrust::transform(rmm::exec_policy(stream)->on(stream),
                    thrust::make_counting_iterator<size_type>(0),
                    thrust::make_counting_iterator<size_type>(strings.size()),
                    d_results,
                    [d_column] __device__(size_type idx) {
                      if (d_column.is_null(idx)) return false;
                      return string::is_integer(d_column.element<string_view>(idx));
                    });
  results->set_null_count(strings.null_count());
  return results;
}

bool all_integer(strings_column_view const& strings, hipStream_t stream = 0)
{
  auto strings_column  = column_device_view::create(strings.parent(), stream);
  auto d_column        = *strings_column;
  auto transformer_itr = thrust::make_transform_iterator(
    thrust::make_counting_iterator<size_type>(0), [d_column] __device__(size_type idx) {
      if (d_column.is_null(idx)) return false;
      return string::is_integer(d_column.element<string_view>(idx));
    });
  return thrust::all_of(rmm::exec_policy(stream)->on(stream),
                        transformer_itr,
                        transformer_itr + strings.size(),
                        thrust::identity<bool>());
}

std::unique_ptr<column> is_float(
  strings_column_view const& strings,
  rmm::mr::device_memory_resource* mr = rmm::mr::get_current_device_resource(),
  hipStream_t stream                 = 0)
{
  auto strings_column = column_device_view::create(strings.parent(), stream);
  auto d_column       = *strings_column;
  // create output column
  auto results = make_numeric_column(
    data_type{type_id::BOOL8},
    strings.size(),
    cudf::detail::copy_bitmask(strings.parent(), rmm::cuda_stream_view{stream}, mr),
    strings.null_count(),
    stream,
    mr);
  auto d_results = results->mutable_view().data<bool>();
  // check strings for valid float chars
  thrust::transform(rmm::exec_policy(stream)->on(stream),
                    thrust::make_counting_iterator<size_type>(0),
                    thrust::make_counting_iterator<size_type>(strings.size()),
                    d_results,
                    [d_column] __device__(size_type idx) {
                      if (d_column.is_null(idx)) return false;
                      return string::is_float(d_column.element<string_view>(idx));
                    });
  results->set_null_count(strings.null_count());
  return results;
}

bool all_float(strings_column_view const& strings, hipStream_t stream = 0)
{
  auto strings_column  = column_device_view::create(strings.parent(), stream);
  auto d_column        = *strings_column;
  auto transformer_itr = thrust::make_transform_iterator(
    thrust::make_counting_iterator<size_type>(0), [d_column] __device__(size_type idx) {
      if (d_column.is_null(idx)) return false;
      return string::is_float(d_column.element<string_view>(idx));
    });
  return thrust::all_of(rmm::exec_policy(stream)->on(stream),
                        transformer_itr,
                        transformer_itr + strings.size(),
                        thrust::identity<bool>());
}

}  // namespace detail

// external API

std::unique_ptr<column> all_characters_of_type(strings_column_view const& strings,
                                               string_character_types types,
                                               string_character_types verify_types,
                                               rmm::mr::device_memory_resource* mr)
{
  CUDF_FUNC_RANGE();
  return detail::all_characters_of_type(strings, types, verify_types, mr);
}

std::unique_ptr<column> filter_characters_of_type(strings_column_view const& strings,
                                                  string_character_types types_to_remove,
                                                  string_scalar const& replacement,
                                                  string_character_types types_to_keep,
                                                  rmm::mr::device_memory_resource* mr)
{
  CUDF_FUNC_RANGE();
  return detail::filter_characters_of_type(
    strings, types_to_remove, replacement, types_to_keep, 0, mr);
}

std::unique_ptr<column> is_integer(strings_column_view const& strings,
                                   rmm::mr::device_memory_resource* mr)
{
  CUDF_FUNC_RANGE();
  return detail::is_integer(strings, mr);
}

std::unique_ptr<column> is_float(strings_column_view const& strings,
                                 rmm::mr::device_memory_resource* mr)
{
  CUDF_FUNC_RANGE();
  return detail::is_float(strings, mr);
}

bool all_integer(strings_column_view const& strings)
{
  CUDF_FUNC_RANGE();
  return detail::all_integer(strings);
}

bool all_float(strings_column_view const& strings)
{
  CUDF_FUNC_RANGE();
  return detail::all_float(strings);
}

}  // namespace strings
}  // namespace cudf
