/*
 * Copyright (c) 2019-2020, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <cudf/column/column_device_view.cuh>
#include <cudf/column/column_factories.hpp>
#include <cudf/detail/nvtx/ranges.hpp>
#include <cudf/scalar/scalar_factories.hpp>
#include <cudf/strings/detail/utilities.hpp>
#include <cudf/strings/find.hpp>
#include <cudf/strings/string_view.cuh>
#include <cudf/strings/strings_column_view.hpp>
#include <cudf/utilities/error.hpp>

#include <thrust/transform.h>

namespace cudf {
namespace strings {
namespace detail {
namespace {
/**
 * @brief Utility to return integer column indicating the postion of
 * target string within each string in a strings column.
 *
 * Null string entries return corresponding null output column entries.
 *
 * @tparam FindFunction Returns integer character position value given a string and target.
 *
 * @param strings Strings column to search for target.
 * @param target String to search for in each string in the strings column.
 * @param start First character position to start the search.
 * @param stop Last character position (exclusive) to end the search.
 * @param pfn Functor used for locating `target` in each string.
 * @param mr Device memory resource used to allocate the returned column's device memory.
 * @param stream CUDA stream used for device memory operations and kernel launches.
 * @return New integer column with character position values.
 */
template <typename FindFunction>
std::unique_ptr<column> find_fn(strings_column_view const& strings,
                                string_scalar const& target,
                                size_type start,
                                size_type stop,
                                FindFunction& pfn,
                                rmm::mr::device_memory_resource* mr,
                                hipStream_t stream)
{
  CUDF_EXPECTS(target.is_valid(), "Parameter target must be valid.");
  CUDF_EXPECTS(start >= 0, "Parameter start must be positive integer or zero.");
  if ((stop > 0) && (start > stop)) CUDF_FAIL("Parameter start must be less than stop.");
  //
  auto d_target       = string_view(target.data(), target.size());
  auto strings_column = column_device_view::create(strings.parent(), stream);
  auto d_strings      = *strings_column;
  auto strings_count  = strings.size();
  // create output column
  auto results      = make_numeric_column(data_type{type_id::INT32},
                                     strings_count,
                                     copy_bitmask(strings.parent(), stream, mr),
                                     strings.null_count(),
                                     stream,
                                     mr);
  auto results_view = results->mutable_view();
  auto d_results    = results_view.data<int32_t>();
  // set the position values by evaluating the passed function
  thrust::transform(rmm::exec_policy(stream)->on(stream),
                    thrust::make_counting_iterator<size_type>(0),
                    thrust::make_counting_iterator<size_type>(strings_count),
                    d_results,
                    [d_strings, pfn, d_target, start, stop] __device__(size_type idx) {
                      int32_t position = -1;
                      if (!d_strings.is_null(idx))
                        position = static_cast<int32_t>(
                          pfn(d_strings.element<string_view>(idx), d_target, start, stop));
                      return position;
                    });
  results->set_null_count(strings.null_count());
  return results;
}

}  // namespace

std::unique_ptr<column> find(strings_column_view const& strings,
                             string_scalar const& target,
                             size_type start                     = 0,
                             size_type stop                      = -1,
                             rmm::mr::device_memory_resource* mr = rmm::mr::get_default_resource(),
                             hipStream_t stream                 = 0)
{
  auto pfn = [] __device__(
               string_view d_string, string_view d_target, size_type start, size_type stop) {
    size_type length = d_string.length();
    if (d_target.empty()) return start > length ? -1 : start;
    size_type begin = (start > length) ? length : start;
    size_type end   = (stop < 0) || (stop > length) ? length : stop;
    return d_string.find(d_target, begin, end - begin);
  };

  return find_fn(strings, target, start, stop, pfn, mr, stream);
}

std::unique_ptr<column> rfind(strings_column_view const& strings,
                              string_scalar const& target,
                              size_type start                     = 0,
                              size_type stop                      = -1,
                              rmm::mr::device_memory_resource* mr = rmm::mr::get_default_resource(),
                              hipStream_t stream                 = 0)
{
  auto pfn = [] __device__(
               string_view d_string, string_view d_target, size_type start, size_type stop) {
    size_type length = d_string.length();
    size_type begin  = (start > length) ? length : start;
    size_type end    = (stop < 0) || (stop > length) ? length : stop;
    if (d_target.empty()) return start > length ? -1 : end;
    return d_string.rfind(d_target, begin, end - begin);
  };

  return find_fn(strings, target, start, stop, pfn, mr, stream);
}

}  // namespace detail

// external APIs

std::unique_ptr<column> find(strings_column_view const& strings,
                             string_scalar const& target,
                             size_type start,
                             size_type stop,
                             rmm::mr::device_memory_resource* mr)
{
  CUDF_FUNC_RANGE();
  return detail::find(strings, target, start, stop, mr);
}

std::unique_ptr<column> rfind(strings_column_view const& strings,
                              string_scalar const& target,
                              size_type start,
                              size_type stop,
                              rmm::mr::device_memory_resource* mr)
{
  CUDF_FUNC_RANGE();
  return detail::rfind(strings, target, start, stop, mr);
}

namespace detail {
namespace {
/**
 * @brief Utility to return a bool column indicating the presence of
 * a given target string in a strings column.
 *
 * Null string entries return corresponding null output column entries.
 *
 * @tparam BoolFunction Return bool value given two strings.
 *
 * @param strings Column of strings to check for target.
 * @param target UTF-8 encoded string to check in strings column.
 * @param pfn Returns bool value if target is found in the given string.
 * @param mr Device memory resource used to allocate the returned column's device memory.
 * @param stream CUDA stream used for device memory operations and kernel launches.
 * @return New BOOL column.
 */
template <typename BoolFunction>
std::unique_ptr<column> contains_fn(strings_column_view const& strings,
                                    string_scalar const& target,
                                    BoolFunction pfn,
                                    rmm::mr::device_memory_resource* mr,
                                    hipStream_t stream)
{
  auto strings_count = strings.size();
  if (strings_count == 0) return make_empty_column(data_type{type_id::BOOL8});

  CUDF_EXPECTS(target.is_valid(), "Parameter target must be valid.");
  if (target.size() == 0)  // empty target string returns true
  {
    auto const true_scalar = make_fixed_width_scalar<bool>(true, stream);
    auto results           = make_column_from_scalar(*true_scalar, strings.size(), mr, stream);
    results->set_null_mask(copy_bitmask(strings.parent(), stream, mr), strings.null_count());
    return results;
  }

  auto d_target       = string_view(target.data(), target.size());
  auto strings_column = column_device_view::create(strings.parent(), stream);
  auto d_strings      = *strings_column;
  // create output column
  auto results      = make_numeric_column(data_type{type_id::BOOL8},
                                     strings_count,
                                     copy_bitmask(strings.parent(), stream, mr),
                                     strings.null_count(),
                                     stream,
                                     mr);
  auto results_view = results->mutable_view();
  auto d_results    = results_view.data<bool>();
  // set the bool values by evaluating the passed function
  thrust::transform(rmm::exec_policy(stream)->on(stream),
                    thrust::make_counting_iterator<size_type>(0),
                    thrust::make_counting_iterator<size_type>(strings_count),
                    d_results,
                    [d_strings, pfn, d_target] __device__(size_type idx) {
                      if (!d_strings.is_null(idx))
                        return bool{pfn(d_strings.element<string_view>(idx), d_target)};
                      return false;
                    });
  results->set_null_count(strings.null_count());
  return results;
}

/**
 * @brief Utility to return a bool column indicating the presence of
 * a string targets[i] in strings[i].
 *
 * Null string entries return corresponding null output column entries.
 *
 * @tparam BoolFunction Return bool value given two strings.
 *
 * @param strings Column of strings to check for `targets[i]`.
 * @param targets Column of strings to be checked in `strings[i]``.
 * @param pfn Returns bool value if target is found in the given string.
 * @param mr Device memory resource used to allocate the returned column's device memory.
 * @param stream CUDA stream used for device memory operations and kernel launches.
 * @return New BOOL column.
 */
template <typename BoolFunction>
std::unique_ptr<column> contains_fn(strings_column_view const& strings,
                                    strings_column_view const& targets,
                                    BoolFunction pfn,
                                    rmm::mr::device_memory_resource* mr,
                                    hipStream_t stream)
{
  auto strings_count = strings.size();
  if (strings_count == 0) return make_empty_column(data_type{type_id::BOOL8});

  auto targets_count = targets.size();
  CUDF_EXPECTS(targets_count > 0, "Must include at least one search target");

  auto targets_column = column_device_view::create(targets.parent(), stream);
  auto d_targets      = *targets_column;
  auto strings_column = column_device_view::create(strings.parent(), stream);
  auto d_strings      = *strings_column;
  // create output column
  auto results      = make_numeric_column(data_type{type_id::BOOL8},
                                     strings_count,
                                     copy_bitmask(strings.parent(), stream, mr),
                                     strings.null_count(),
                                     stream,
                                     mr);
  auto results_view = results->mutable_view();
  auto d_results    = results_view.data<bool>();
  // set the bool values by evaluating the passed function
  thrust::transform(
    rmm::exec_policy(stream)->on(stream),
    thrust::make_counting_iterator<size_type>(0),
    thrust::make_counting_iterator<size_type>(strings_count),
    d_results,
    [d_strings, pfn, d_targets] __device__(size_type idx) {
      // empty target string returns true
      if (d_targets.is_valid(idx) && d_targets.element<string_view>(idx).length() == 0) {
        return true;
      } else if (!d_strings.is_null(idx) && !d_targets.is_null(idx)) {
        return bool{pfn(d_strings.element<string_view>(idx), d_targets.element<string_view>(idx))};
      } else {
        return false;
      }
    });
  results->set_null_count(strings.null_count());
  return results;
}

}  // namespace

std::unique_ptr<column> contains(
  strings_column_view const& strings,
  string_scalar const& target,
  rmm::mr::device_memory_resource* mr = rmm::mr::get_default_resource(),
  hipStream_t stream                 = 0)
{
  auto pfn = [] __device__(string_view d_string, string_view d_target) {
    return d_string.find(d_target) >= 0;
  };
  return contains_fn(strings, target, pfn, mr, stream);
}

std::unique_ptr<column> starts_with(
  strings_column_view const& strings,
  string_scalar const& target,
  rmm::mr::device_memory_resource* mr = rmm::mr::get_default_resource(),
  hipStream_t stream                 = 0)
{
  auto pfn = [] __device__(string_view d_string, string_view d_target) {
    return d_string.find(d_target) == 0;
  };
  return contains_fn(strings, target, pfn, mr, stream);
}

std::unique_ptr<column> starts_with(
  strings_column_view const& strings,
  strings_column_view const& targets,
  rmm::mr::device_memory_resource* mr = rmm::mr::get_default_resource(),
  hipStream_t stream                 = 0)
{
  auto pfn = [] __device__(string_view d_string, string_view d_target) {
    return d_string.find(d_target) == 0;
  };
  return contains_fn(strings, targets, pfn, mr, stream);
}

std::unique_ptr<column> ends_with(
  strings_column_view const& strings,
  string_scalar const& target,
  rmm::mr::device_memory_resource* mr = rmm::mr::get_default_resource(),
  hipStream_t stream                 = 0)
{
  auto pfn = [] __device__(string_view d_string, string_view d_target) {
    auto str_length = d_string.length();
    auto tgt_length = d_target.length();
    if (str_length < tgt_length) return false;
    return d_string.find(d_target, str_length - tgt_length) >= 0;
  };

  return contains_fn(strings, target, pfn, mr, stream);
}

std::unique_ptr<column> ends_with(
  strings_column_view const& strings,
  strings_column_view const& targets,
  rmm::mr::device_memory_resource* mr = rmm::mr::get_default_resource(),
  hipStream_t stream                 = 0)
{
  auto pfn = [] __device__(string_view d_string, string_view d_target) {
    auto str_length = d_string.length();
    auto tgt_length = d_target.length();
    if (str_length < tgt_length) return false;
    return d_string.find(d_target, str_length - tgt_length) >= 0;
  };

  return contains_fn(strings, targets, pfn, mr, stream);
}

}  // namespace detail

// external APIs

std::unique_ptr<column> contains(strings_column_view const& strings,
                                 string_scalar const& target,
                                 rmm::mr::device_memory_resource* mr)
{
  CUDF_FUNC_RANGE();
  return detail::contains(strings, target, mr);
}

std::unique_ptr<column> starts_with(strings_column_view const& strings,
                                    string_scalar const& target,
                                    rmm::mr::device_memory_resource* mr)
{
  CUDF_FUNC_RANGE();
  return detail::starts_with(strings, target, mr);
}

std::unique_ptr<column> starts_with(strings_column_view const& strings,
                                    strings_column_view const& targets,
                                    rmm::mr::device_memory_resource* mr)
{
  CUDF_FUNC_RANGE();
  return detail::starts_with(strings, targets, mr);
}

std::unique_ptr<column> ends_with(strings_column_view const& strings,
                                  string_scalar const& target,
                                  rmm::mr::device_memory_resource* mr)
{
  CUDF_FUNC_RANGE();
  return detail::ends_with(strings, target, mr);
}

std::unique_ptr<column> ends_with(strings_column_view const& strings,
                                  strings_column_view const& targets,
                                  rmm::mr::device_memory_resource* mr)
{
  CUDF_FUNC_RANGE();
  return detail::ends_with(strings, targets, mr);
}

}  // namespace strings
}  // namespace cudf
