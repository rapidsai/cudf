/*
 * Copyright (c) 2019, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <cudf/column/column_factories.hpp>
#include <cudf/column/column_device_view.cuh>
#include <cudf/strings/find.hpp>
#include <cudf/strings/strings_column_view.hpp>
#include <cudf/strings/string_view.cuh>
#include <cudf/utilities/error.hpp>
#include "./utilities.hpp"

#include <thrust/transform.h>

namespace cudf
{
namespace strings
{
namespace detail
{
namespace
{

/**
 * @brief Utility to return integer column indicating the postion of
 * target string within each string in a strings column.
 *
 * Null string entries return corresponding null output column entries.
 *
 * @tparam FindFunction Returns integer character position value given a string and target.
 *
 * @param strings Strings column to search for target.
 * @param target String to search for in each string in the strings column.
 * @param start First character position to start the search.
 * @param stop Last character position (exclusive) to end the search.
 * @param pfn Strings instance for this operation.
 * @param mr Resource for allocating device memory.
 * @param stream Stream to use for kernel calls.
 * @return New integer column with character position values.
 */
template <typename FindFunction>
std::unique_ptr<column> find_fn( strings_column_view const& strings,
                                 string_scalar const& target,
                                 size_type start, size_type stop,
                                 FindFunction& pfn,
                                 rmm::mr::device_memory_resource* mr,
                                 hipStream_t stream )
{
    CUDF_EXPECTS( target.is_valid() && target.size()>0, "Parameter target must not be empty.");
    CUDF_EXPECTS( start >= 0, "Parameter start must be positive integer or zero.");
    if( (stop) > 0 && (start >stop) )
        CUDF_FAIL( "Parameter start must be less than stop.");
    //
    auto d_target = string_view(target.data(),target.size());
    auto strings_column = column_device_view::create(strings.parent(),stream);
    auto d_strings = *strings_column;
    auto strings_count = strings.size();
    // create output column
    auto results = make_numeric_column( data_type{INT32}, strings_count,
        copy_bitmask( strings.parent(), stream, mr ), strings.null_count(), stream, mr);
    auto results_view = results->mutable_view();
    auto d_results = results_view.data<int32_t>();
    // set the position values by evaluating the passed function
    thrust::transform( rmm::exec_policy(stream)->on(stream),
        thrust::make_counting_iterator<size_type>(0),
        thrust::make_counting_iterator<size_type>(strings_count),
        d_results, [d_strings, pfn, d_target, start, stop] __device__ (size_type idx) {
            int32_t position = -1;
            if( !d_strings.is_null(idx) )
                position = static_cast<int32_t>(pfn(d_strings.element<string_view>(idx),d_target,start,stop));
            return position;
        });
    results->set_null_count(strings.null_count());
    return results;
}

} // namespace

std::unique_ptr<column> find( strings_column_view const& strings,
                              string_scalar const& target,
                              size_type start=0, size_type stop=-1,
                              rmm::mr::device_memory_resource* mr = rmm::mr::get_default_resource(),
                              hipStream_t stream=0 )
{
    auto pfn = [] __device__ (string_view d_string, string_view d_target,
                              size_type start, size_type stop) {
        size_type length = d_string.length();
        size_type end = (stop < 0) || (stop > length) ? length : stop;
        return d_string.find( d_target, start, end-start );
    };

    return find_fn( strings, target, start, stop, pfn, mr, stream);
}

std::unique_ptr<column> rfind( strings_column_view const& strings,
                               string_scalar const& target,
                               size_type start=0, size_type stop=-1,
                               rmm::mr::device_memory_resource* mr = rmm::mr::get_default_resource(),
                               hipStream_t stream=0 )
{
    auto pfn = [] __device__ (string_view d_string, string_view d_target,
                              size_type start, size_type stop) {
        size_type length = d_string.length();
        size_type end = (stop < 0) || (stop > length) ? length : stop;
        return d_string.rfind( d_target, start, end-start );
    };

    return find_fn( strings, target, start, stop, pfn, mr, stream);
}

} // namespace detail

// external APIs

std::unique_ptr<column> find( strings_column_view const& strings,
                              string_scalar const& target,
                              size_type start, size_type stop,
                              rmm::mr::device_memory_resource* mr)
{
    return detail::find( strings, target, start, stop, mr );
}

std::unique_ptr<column> rfind( strings_column_view const& strings,
                               string_scalar const& target,
                               size_type start, size_type stop,
                               rmm::mr::device_memory_resource* mr)
{
    return detail::rfind( strings, target, start, stop, mr );
}

namespace detail
{
namespace
{

/**
 * @brief Utility to return a bool column indicating the presence of
 * a given target string in a strings column.
 *
 * Null string entries return corresponding null output column entries.
 *
 * @tparam BoolFunction Return bool value given two strings.
 *
 * @param strings Column of strings to check for target.
 * @param target UTF-8 encoded string to check in strings column.
 * @param pfn Returns bool value if target is found in the given string.
 * @param mr Resource for allocating device memory.
 * @param stream Stream to use for kernel calls.
 * @return New BOOL8 column.
 */
template <typename BoolFunction>
std::unique_ptr<column> contains_fn( strings_column_view const& strings,
                                     string_scalar const& target,
                                     BoolFunction pfn,
                                     rmm::mr::device_memory_resource* mr,
                                     hipStream_t stream )
{
    auto strings_count = strings.size();
    if( strings_count == 0 )
        return make_numeric_column( data_type{BOOL8}, 0 );

    CUDF_EXPECTS( target.is_valid() && target.size()>0, "Parameter target must not be empty.");
    auto d_target = string_view( target.data(), target.size());
    auto strings_column = column_device_view::create(strings.parent(),stream);
    auto d_strings = *strings_column;
    // create output column
    auto results = make_numeric_column( data_type{BOOL8}, strings_count,
        copy_bitmask( strings.parent(), stream, mr ), strings.null_count(), stream, mr);
    auto results_view = results->mutable_view();
    auto d_results = results_view.data<experimental::bool8>();
    // set the bool values but evaluating the passed function
    thrust::transform( rmm::exec_policy(stream)->on(stream),
        thrust::make_counting_iterator<size_type>(0),
        thrust::make_counting_iterator<size_type>(strings_count),
        d_results,
        [d_strings, pfn, d_target] __device__ (size_type idx) {
            experimental::bool8 result = 0;
            if( !d_strings.is_null(idx) )
                result = static_cast<experimental::bool8>(pfn(d_strings.element<string_view>(idx), d_target));
            return result;
        });
    results->set_null_count(strings.null_count());
    return results;
}

} // namespace


std::unique_ptr<column> contains( strings_column_view const& strings,
                                  string_scalar const& target,
                                  rmm::mr::device_memory_resource* mr = rmm::mr::get_default_resource(),
                                  hipStream_t stream=0 )
{
    auto pfn = [] __device__ (string_view d_string, string_view d_target) {
        return d_string.find( d_target )>=0;
    };

    return contains_fn( strings, target, pfn, mr, stream );
}

std::unique_ptr<column> starts_with( strings_column_view const& strings,
                                     string_scalar const& target,
                                     rmm::mr::device_memory_resource* mr = rmm::mr::get_default_resource(),
                                     hipStream_t stream=0 )
{
    auto pfn = [] __device__ (string_view d_string, string_view d_target) {
        return d_string.find( d_target )==0;
    };
    return contains_fn( strings, target, pfn, mr, stream );
}

std::unique_ptr<column> ends_with( strings_column_view const& strings,
                                   string_scalar const& target,
                                   rmm::mr::device_memory_resource* mr = rmm::mr::get_default_resource(),
                                   hipStream_t stream=0 )
{

    auto pfn = [] __device__ (string_view d_string, string_view d_target) {
        auto str_length = d_string.length();
        auto tgt_length = d_target.length();
        if( str_length <= tgt_length )
            return false;
        return d_string.find( d_target, str_length - tgt_length )>=0;
    };

    return contains_fn( strings, target, pfn, mr, stream );
}

} // namespace detail

// external APIs

std::unique_ptr<column> contains( strings_column_view const& strings,
                                  string_scalar const& target,
                                  rmm::mr::device_memory_resource* mr )
{
    return detail::contains( strings, target, mr );
}

std::unique_ptr<column> starts_with( strings_column_view const& strings,
                                     string_scalar const& target,
                                     rmm::mr::device_memory_resource* mr )
{
    return detail::starts_with( strings, target, mr );
}

std::unique_ptr<column> ends_with( strings_column_view const& strings,
                                   string_scalar const& target,
                                   rmm::mr::device_memory_resource* mr )
{
    return detail::ends_with( strings, target, mr );
}

} // namespace strings
} // namespace cudf
