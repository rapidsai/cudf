/*
 * Copyright (c) 2019-2022, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <strings/count_matches.hpp>
#include <strings/regex/regex_program_impl.h>
#include <strings/regex/utilities.cuh>

#include <cudf/column/column.hpp>
#include <cudf/column/column_device_view.cuh>
#include <cudf/detail/get_value.cuh>
#include <cudf/detail/null_mask.hpp>
#include <cudf/detail/nvtx/ranges.hpp>
#include <cudf/strings/detail/strings_column_factories.cuh>
#include <cudf/strings/detail/utilities.hpp>
#include <cudf/strings/findall.hpp>
#include <cudf/strings/string_view.cuh>
#include <cudf/strings/strings_column_view.hpp>
#include <cudf/utilities/default_stream.hpp>

#include <rmm/cuda_stream_view.hpp>
#include <rmm/exec_policy.hpp>

#include <thrust/pair.h>
#include <thrust/scan.h>

namespace cudf {
namespace strings {
namespace detail {

using string_index_pair = thrust::pair<const char*, size_type>;

namespace {

/**
 * @brief This functor handles extracting matched strings by applying the compiled regex pattern
 * and creating string_index_pairs for all the substrings.
 */
struct findall_fn {
  column_device_view const d_strings;
  offset_type const* d_offsets;
  string_index_pair* d_indices;

  __device__ void operator()(size_type const idx, reprog_device const prog, int32_t const prog_idx)
  {
    if (d_strings.is_null(idx)) { return; }
    auto const d_str  = d_strings.element<string_view>(idx);
    auto const nchars = d_str.length();

    auto d_output        = d_indices + d_offsets[idx];
    size_type output_idx = 0;

    size_type begin = 0;
    size_type end   = nchars;
    while ((begin < end) && (prog.find(prog_idx, d_str, begin, end) > 0)) {
      auto const spos = d_str.byte_offset(begin);  // convert
      auto const epos = d_str.byte_offset(end);    // to bytes

      d_output[output_idx++] = string_index_pair{d_str.data() + spos, (epos - spos)};

      begin = end + (begin == end);
      end   = nchars;
    }
  }
};

std::unique_ptr<column> findall_util(column_device_view const& d_strings,
                                     reprog_device& d_prog,
                                     size_type total_matches,
                                     offset_type const* d_offsets,
                                     rmm::cuda_stream_view stream,
                                     rmm::mr::device_memory_resource* mr)
{
  rmm::device_uvector<string_index_pair> indices(total_matches, stream);

  launch_for_each_kernel(
    findall_fn{d_strings, d_offsets, indices.data()}, d_prog, d_strings.size(), stream);

  return make_strings_column(indices.begin(), indices.end(), stream, mr);
}

}  // namespace

//
std::unique_ptr<column> findall(strings_column_view const& input,
                                regex_program const& prog,
                                rmm::cuda_stream_view stream,
                                rmm::mr::device_memory_resource* mr)
{
  auto const strings_count = input.size();
  auto const d_strings     = column_device_view::create(input.parent(), stream);

  // create device object from regex_program
  auto d_prog = regex_device_builder::create_prog_device(prog, stream);

  // Create lists offsets column
  auto offsets   = count_matches(*d_strings, *d_prog, strings_count + 1, stream, mr);
  auto d_offsets = offsets->mutable_view().data<offset_type>();

  // Convert counts into offsets
  thrust::exclusive_scan(
    rmm::exec_policy(stream), d_offsets, d_offsets + strings_count + 1, d_offsets);

  // Create indices vector with the total number of groups that will be extracted
  auto const total_matches =
    cudf::detail::get_value<size_type>(offsets->view(), strings_count, stream);

  auto strings_output = findall_util(*d_strings, *d_prog, total_matches, d_offsets, stream, mr);

  // Build the lists column from the offsets and the strings
  return make_lists_column(strings_count,
                           std::move(offsets),
                           std::move(strings_output),
                           input.null_count(),
                           cudf::detail::copy_bitmask(input.parent(), stream, mr),
                           stream,
                           mr);
}

}  // namespace detail

// external API

std::unique_ptr<column> findall(strings_column_view const& input,
                                std::string_view pattern,
                                regex_flags const flags,
                                rmm::mr::device_memory_resource* mr)
{
  CUDF_FUNC_RANGE();
  auto const h_prog = regex_program::create(pattern, flags, capture_groups::NON_CAPTURE);
  return detail::findall(input, *h_prog, cudf::get_default_stream(), mr);
}

std::unique_ptr<column> findall(strings_column_view const& input,
                                regex_program const& prog,
                                rmm::mr::device_memory_resource* mr)
{
  CUDF_FUNC_RANGE();
  return detail::findall(input, prog, cudf::get_default_stream(), mr);
}

}  // namespace strings
}  // namespace cudf
