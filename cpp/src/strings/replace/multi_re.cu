#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019-2020, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <strings/regex/regex.cuh>
#include <strings/utilities.cuh>
#include <strings/utilities.hpp>

#include <cudf/column/column.hpp>
#include <cudf/column/column_device_view.cuh>
#include <cudf/column/column_factories.hpp>
#include <cudf/detail/nvtx/ranges.hpp>
#include <cudf/null_mask.hpp>
#include <cudf/strings/detail/utilities.hpp>
#include <cudf/strings/replace_re.hpp>
#include <cudf/strings/string_view.cuh>
#include <cudf/strings/strings_column_view.hpp>

#include <rmm/cuda_stream_view.hpp>

namespace cudf {
namespace strings {
namespace detail {
namespace {
// this is a [begin,end) pair of character positions when a substring is matched
using found_range = thrust::pair<size_type, size_type>;

/**
 * @brief This functor handles replacing strings by applying the compiled regex patterns
 * and inserting the corresponding new string within the matched range of characters.
 *
 * The logic includes computing the size of each string and also writing the output.
 *
 * The stack is used to keep progress on evaluating the regex instructions on each string.
 * So the size of the stack is in proportion to the number of instructions in the given regex
 * pattern.
 *
 * There are three call types based on the number of regex instructions in the given pattern.
 * Small to medium instruction lengths can use the stack effectively though smaller executes faster.
 * Longer patterns require global memory. Shorter patterns are common in data cleaning.
 */
template <size_t stack_size>
struct replace_multi_regex_fn {
  column_device_view const d_strings;
  reprog_device* progs;  // array of regex progs
  size_type number_of_patterns;
  found_range* d_found_ranges;       // working array matched (begin,end) values
  column_device_view const d_repls;  // replacment strings
  int32_t* d_offsets{};              // these are null when
  char* d_chars{};                   // only computing size

  __device__ void operator()(size_type idx)
  {
    if (d_strings.is_null(idx)) {
      if (!d_chars) d_offsets[idx] = 0;
      return;
    }
    u_char data1[stack_size];
    u_char data2[stack_size];
    auto const d_str      = d_strings.element<string_view>(idx);
    auto const nchars     = d_str.length();      // number of characters in input string
    auto nbytes           = d_str.size_bytes();  // number of bytes in input string
    auto in_ptr           = d_str.data();        // input pointer
    auto out_ptr          = d_chars ? d_chars + d_offsets[idx] : nullptr;
    found_range* d_ranges = d_found_ranges + (idx * number_of_patterns);
    size_type lpos        = 0;
    size_type ch_pos      = 0;
    // initialize the working ranges memory to -1's
    thrust::fill(thrust::seq, d_ranges, d_ranges + number_of_patterns, found_range{-1, 1});
    // process string one character at a time
    while (ch_pos < nchars) {
      // this minimizes the regex-find calls by only calling it for stale patterns
      // -- those that have not previously matched up to this point (ch_pos)
      for (size_type ptn_idx = 0; ptn_idx < number_of_patterns; ++ptn_idx) {
        if (d_ranges[ptn_idx].first >= ch_pos)  // previously matched here
          continue;                             // or later in the string
        reprog_device prog = progs[ptn_idx];
        prog.set_stack_mem(data1, data2);
        auto begin = static_cast<int32_t>(ch_pos);
        auto end   = static_cast<int32_t>(nchars);
        if (!prog.is_empty() && prog.find(idx, d_str, begin, end) > 0)
          d_ranges[ptn_idx] = found_range{begin, end};  // found a match
        else
          d_ranges[ptn_idx] = found_range{nchars, nchars};  // this pattern is done
      }
      // all the ranges have been updated from each regex match;
      // look for any that match at this character position (ch_pos)
      auto itr = thrust::find_if(
        thrust::seq, d_ranges, d_ranges + number_of_patterns, [ch_pos] __device__(auto range) {
          return range.first == ch_pos;
        });
      if (itr !=
          d_ranges +
            number_of_patterns) {  // match found, compute and replace the string in the output
        size_type ptn_idx  = static_cast<size_type>(itr - d_ranges);
        size_type begin    = d_ranges[ptn_idx].first;
        size_type end      = d_ranges[ptn_idx].second;
        string_view d_repl = d_repls.size() > 1 ? d_repls.element<string_view>(ptn_idx)
                                                : d_repls.element<string_view>(0);
        auto spos = d_str.byte_offset(begin);
        auto epos = d_str.byte_offset(end);
        nbytes += d_repl.size_bytes() - (epos - spos);
        if (out_ptr) {  // copy unmodified content plus new replacement string
          out_ptr = copy_and_increment(out_ptr, in_ptr + lpos, spos - lpos);
          out_ptr = copy_string(out_ptr, d_repl);
          lpos    = epos;
        }
        ch_pos = end - 1;
      }
      ++ch_pos;
    }
    if (out_ptr)  // copy the remainder
      memcpy(out_ptr, in_ptr + lpos, d_str.size_bytes() - lpos);
    else
      d_offsets[idx] = static_cast<int32_t>(nbytes);
  }
};

}  // namespace

std::unique_ptr<column> replace_re(
  strings_column_view const& strings,
  std::vector<std::string> const& patterns,
  strings_column_view const& repls,
  rmm::cuda_stream_view stream,
  rmm::mr::device_memory_resource* mr = rmm::mr::get_current_device_resource())
{
  auto strings_count = strings.size();
  if (strings_count == 0) return make_empty_strings_column(stream, mr);
  if (patterns.empty())  // no patterns; just return a copy
    return std::make_unique<column>(strings.parent());

  CUDF_EXPECTS(!repls.has_nulls(), "Parameter repls must not have any nulls");

  auto strings_column = column_device_view::create(strings.parent(), stream);
  auto d_strings      = *strings_column;
  auto repls_column   = column_device_view::create(repls.parent(), stream);
  auto d_repls        = *repls_column;
  auto d_flags        = get_character_flags_table();
  // compile regexes into device objects
  size_type regex_insts = 0;
  std::vector<std::unique_ptr<reprog_device, std::function<void(reprog_device*)>>> h_progs;
  rmm::device_vector<reprog_device> progs;
  for (auto itr = patterns.begin(); itr != patterns.end(); ++itr) {
    auto prog  = reprog_device::create(*itr, d_flags, strings_count, stream);
    auto insts = prog->insts_counts();
    if (insts > regex_insts) regex_insts = insts;
    progs.push_back(*prog);
    h_progs.emplace_back(std::move(prog));
  }
  auto d_progs = progs.data().get();

  // copy null mask
  auto null_mask  = copy_bitmask(strings.parent());
  auto null_count = strings.null_count();
  // create working buffer for ranges pairs
  rmm::device_vector<found_range> found_ranges(patterns.size() * strings_count);
  auto d_found_ranges = found_ranges.data().get();

  // create child columns
  std::pair<std::unique_ptr<column>, std::unique_ptr<column>> children(nullptr, nullptr);
  // Each invocation is predicated on the stack size which is dependent on the number of regex
  // instructions
  if ((regex_insts > MAX_STACK_INSTS) || (regex_insts <= RX_SMALL_INSTS))
    children = make_strings_children(
      replace_multi_regex_fn<RX_STACK_SMALL>{
        d_strings, d_progs, static_cast<size_type>(progs.size()), d_found_ranges, d_repls},
      strings_count,
      null_count,
      stream,
      mr);
  else if (regex_insts <= RX_MEDIUM_INSTS)
    children = make_strings_children(
      replace_multi_regex_fn<RX_STACK_MEDIUM>{
        d_strings, d_progs, static_cast<size_type>(progs.size()), d_found_ranges, d_repls},
      strings_count,
      null_count,
      stream,
      mr);
  else
    children = make_strings_children(
      replace_multi_regex_fn<RX_STACK_LARGE>{
        d_strings, d_progs, static_cast<size_type>(progs.size()), d_found_ranges, d_repls},
      strings_count,
      null_count,
      stream,
      mr);

  return make_strings_column(strings_count,
                             std::move(children.first),
                             std::move(children.second),
                             null_count,
                             std::move(null_mask),
                             stream,
                             mr);
}

}  // namespace detail

// external API

std::unique_ptr<column> replace_re(strings_column_view const& strings,
                                   std::vector<std::string> const& patterns,
                                   strings_column_view const& repls,
                                   rmm::mr::device_memory_resource* mr)
{
  CUDF_FUNC_RANGE();
  return detail::replace_re(strings, patterns, repls, rmm::cuda_stream_default, mr);
}

}  // namespace strings
}  // namespace cudf
