#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2020, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <cudf/column/column.hpp>
#include <cudf/column/column_device_view.cuh>
#include <cudf/column/column_factories.hpp>
#include <cudf/detail/nvtx/ranges.hpp>
#include <cudf/null_mask.hpp>
#include <cudf/strings/detail/utilities.hpp>
#include <cudf/strings/replace_re.hpp>
#include <cudf/strings/string_view.cuh>
#include <cudf/strings/strings_column_view.hpp>
#include <strings/regex/regex.cuh>
#include <strings/utilities.cuh>
#include <strings/utilities.hpp>

#include <regex>

namespace cudf {
namespace strings {
namespace detail {
namespace {
using backref_type = thrust::pair<size_type, size_type>;

/**
 * @brief Parse the back-ref index and position values from a given replace format.
 *
 * The backref numbers are expected to be 1-based.
 *
 * Returns a modified string without back-ref indicators.
 * ```
 * Example:
 *    for input string:    'hello \2 and \1'
 *    the returned pairs:  (2,6),(1,11)
 *    returned string is:  'hello  and '
 * ```
 */
std::string parse_backrefs(std::string const& repl, std::vector<backref_type>& backrefs)
{
  std::string str = repl;  // make a modifiable copy
  std::smatch m;
  std::regex ex("(\\\\\\d+)");  // this searches for backslash-number(s); example "\1"
  std::string rtn;              // result without refs
  size_type byte_offset = 0;
  while (std::regex_search(str, m, ex)) {
    if (m.size() == 0) break;
    backref_type item;
    std::string bref   = m[0];
    size_type position = static_cast<size_type>(m.position(0));
    size_type length   = static_cast<size_type>(bref.length());
    byte_offset += position;
    item.first = std::atoi(bref.c_str() + 1);  // back-ref index number
    CUDF_EXPECTS(item.first > 0, "Back-reference numbers must be greater than 0");
    item.second = byte_offset;  // position within the string
    rtn += str.substr(0, position);
    str = str.substr(position + length);
    backrefs.push_back(item);
  }
  if (!str.empty())  // add the remainder
    rtn += str;      // of the string
  return rtn;
}

/**
 * @brief This functor handles replacing strings by applying the compiled regex pattern
 * and inserting the at the backref position indicated in the replacement template.
 *
 * The logic includes computing the size of each string and also writing the output.
 *
 * The stack is used to keep progress on evaluating the regex instructions on each string.
 * So the size of the stack is in proportion to the number of instructions in the given regex
 * pattern.
 *
 * There are three call types based on the number of regex instructions in the given pattern.
 * Small to medium instruction lengths can use the stack effectively though smaller executes faster.
 * Longer patterns require global memory. Shorter patterns are common in data cleaning.
 *
 */
template <size_t stack_size>
struct backrefs_fn {
  column_device_view const d_strings;
  reprog_device prog;
  string_view const d_repl;  // string replacement template
  rmm::device_vector<backref_type>::iterator backrefs_begin;
  rmm::device_vector<backref_type>::iterator backrefs_end;
  const int32_t* d_offsets{};  // these are null when
  char* d_chars{};             // only computing size

  __device__ size_type operator()(size_type idx)
  {
    if (d_strings.is_null(idx)) return 0;
    u_char data1[stack_size];
    u_char data2[stack_size];
    prog.set_stack_mem(data1, data2);
    string_view const d_str = d_strings.element<string_view>(idx);
    auto const nchars       = d_str.length();      // number of characters in input string
    auto nbytes             = d_str.size_bytes();  // number of bytes in input string
    const char* in_ptr      = d_str.data();
    char* out_ptr           = d_offsets ? (d_chars + d_offsets[idx]) : nullptr;
    size_type lpos          = 0;       // last byte position processed in d_str
    size_type begin         = 0;       // first character position matching regex
    size_type end           = nchars;  // last character position (exclusive)
    // copy input to output replacing strings as we go
    while (prog.find(idx, d_str, begin, end) > 0)  // inits the begin/end vars
    {
      auto spos = d_str.byte_offset(begin);           // get offset for these
      auto epos = d_str.byte_offset(end);             // character position values
      nbytes += d_repl.size_bytes() - (epos - spos);  // compute new size
      if (out_ptr) out_ptr = copy_and_increment(out_ptr, in_ptr + lpos, spos - lpos);
      size_type lpos_template = 0;              // last end pos of replace template
      auto const repl_ptr     = d_repl.data();  // replace template pattern
      thrust::for_each(
        thrust::seq, backrefs_begin, backrefs_end, [&] __device__(backref_type backref) {
          if (out_ptr) {
            auto const copy_length = backref.second - lpos_template;
            out_ptr = copy_and_increment(out_ptr, repl_ptr + lpos_template, copy_length);
            lpos_template += copy_length;
          }
          // extract the specific group's string for this backref's index
          size_type spos_extract = begin;  // these are modified
          size_type epos_extract = end;    // by extract()
          if ((prog.extract(idx, d_str, spos_extract, epos_extract, backref.first - 1) <= 0) ||
              (epos_extract <= spos_extract))
            return;  // no value for this backref number; that is ok
          spos_extract = d_str.byte_offset(spos_extract);  // convert
          epos_extract = d_str.byte_offset(epos_extract);  // to bytes
          nbytes += epos_extract - spos_extract;
          if (out_ptr)
            out_ptr =
              copy_and_increment(out_ptr, in_ptr + spos_extract, (epos_extract - spos_extract));
        });
      if (out_ptr && (lpos_template < d_repl.size_bytes()))  // copy remainder of template
        out_ptr = copy_and_increment(
          out_ptr, repl_ptr + lpos_template, d_repl.size_bytes() - lpos_template);
      lpos  = epos;
      begin = end;
      end   = nchars;
    }
    if (out_ptr && (lpos < d_str.size_bytes()))  // copy remainder of input string
      memcpy(out_ptr, in_ptr + lpos, d_str.size_bytes() - lpos);
    return nbytes;
  }
};

}  // namespace

//
std::unique_ptr<column> replace_with_backrefs(
  strings_column_view const& strings,
  std::string const& pattern,
  std::string const& repl,
  rmm::mr::device_memory_resource* mr = rmm::mr::get_default_resource(),
  hipStream_t stream                 = 0)
{
  auto strings_count = strings.size();
  if (strings_count == 0) return make_empty_strings_column(mr, stream);

  CUDF_EXPECTS(!pattern.empty(), "Parameter pattern must not be empty");
  CUDF_EXPECTS(!repl.empty(), "Parameter repl must not be empty");

  auto strings_column = column_device_view::create(strings.parent(), stream);
  auto d_strings      = *strings_column;
  // compile regex into device object
  auto prog   = reprog_device::create(pattern, get_character_flags_table(), strings_count, stream);
  auto d_prog = *prog;
  auto regex_insts = d_prog.insts_counts();

  // parse the repl string for backref indicators
  std::vector<backref_type> h_backrefs;
  std::string repl_template = parse_backrefs(repl, h_backrefs);
  rmm::device_vector<backref_type> backrefs(h_backrefs);
  string_scalar repl_scalar(repl_template);
  string_view d_repl_template{repl_scalar.data(), repl_scalar.size()};

  // copy null mask
  auto null_mask  = copy_bitmask(strings.parent());
  auto null_count = strings.null_count();

  // create child columns
  std::pair<std::unique_ptr<column>, std::unique_ptr<column>> children(nullptr, nullptr);
  // Each invocation is predicated on the stack size which is dependent on the number of regex
  // instructions
  if ((regex_insts > MAX_STACK_INSTS) || (regex_insts <= RX_SMALL_INSTS))
    children = make_strings_children(
      backrefs_fn<RX_STACK_SMALL>{
        d_strings, d_prog, d_repl_template, backrefs.begin(), backrefs.end()},
      strings_count,
      null_count,
      mr,
      stream);
  else if (regex_insts <= RX_MEDIUM_INSTS)
    children = make_strings_children(
      backrefs_fn<RX_STACK_MEDIUM>{
        d_strings, d_prog, d_repl_template, backrefs.begin(), backrefs.end()},
      strings_count,
      null_count,
      mr,
      stream);
  else
    children = make_strings_children(
      backrefs_fn<RX_STACK_LARGE>{
        d_strings, d_prog, d_repl_template, backrefs.begin(), backrefs.end()},
      strings_count,
      null_count,
      mr,
      stream);
  //
  return make_strings_column(strings_count,
                             std::move(children.first),
                             std::move(children.second),
                             null_count,
                             std::move(null_mask),
                             stream,
                             mr);
}

}  // namespace detail

// external API

std::unique_ptr<column> replace_with_backrefs(strings_column_view const& strings,
                                              std::string const& pattern,
                                              std::string const& repl,
                                              rmm::mr::device_memory_resource* mr)
{
  CUDF_FUNC_RANGE();
  return detail::replace_with_backrefs(strings, pattern, repl, mr);
}

}  // namespace strings
}  // namespace cudf
