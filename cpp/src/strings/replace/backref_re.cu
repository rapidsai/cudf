#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2020-2021, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "backref_re.cuh"

#include <strings/regex/regex.cuh>
#include <strings/utilities.hpp>

#include <cudf/column/column.hpp>
#include <cudf/column/column_device_view.cuh>
#include <cudf/column/column_factories.hpp>
#include <cudf/detail/null_mask.hpp>
#include <cudf/detail/nvtx/ranges.hpp>
#include <cudf/strings/replace_re.hpp>
#include <cudf/strings/string_view.cuh>
#include <cudf/strings/strings_column_view.hpp>

#include <rmm/cuda_stream_view.hpp>

#include <regex>

namespace cudf {
namespace strings {
namespace detail {
namespace {

/**
 * @brief Return the capturing group index pattern to use with the given replacement string.
 *
 * Only two patterns are supported at this time `\d` and `${d}` where `d` is an integer in
 * the range 1-99. The `\d` pattern is returned by default unless no `\d` pattern is found in
 * the `repl` string,
 *
 * Reference: https://www.regular-expressions.info/refreplacebackref.html
 */
std::string get_backref_pattern(std::string const& repl)
{
  std::string const backslash_pattern = "\\\\(\\d+)";
  std::string const bracket_pattern   = "\\$\\{(\\d+)\\}";
  std::smatch m;
  return std::regex_search(repl, m, std::regex(backslash_pattern)) ? backslash_pattern
                                                                   : bracket_pattern;
}
/**
 * @brief Parse the back-ref index and position values from a given replace format.
 *
 * The back-ref numbers are expected to be 1-based.
 *
 * Returns a modified string without back-ref indicators and a vector of back-ref
 * byte position pairs. These are used by the device code to build the output
 * string by placing the captured group elements into the replace format.
 *
 * For example, for input string 'hello \2 and \1' the returned `backref_type` vector
 * contains `[(2,6),(1,11)]` and the returned string is 'hello  and '.
 */
std::pair<std::string, std::vector<backref_type>> parse_backrefs(std::string const& repl)
{
  std::vector<backref_type> backrefs;
  std::string str = repl;  // make a modifiable copy
  std::smatch m;
  std::regex ex(get_backref_pattern(repl));
  std::string rtn;
  size_type byte_offset = 0;
  while (std::regex_search(str, m, ex) && !m.empty()) {
    // parse the back-ref index number
    size_type const index = static_cast<size_type>(std::atoi(std::string{m[1]}.c_str()));
    CUDF_EXPECTS(index > 0 && index < 100, "Group index numbers must be in the range 1-99");

    // store the new byte offset and index value
    size_type const position = static_cast<size_type>(m.position(0));
    byte_offset += position;
    backrefs.push_back({index, byte_offset});

    // update the output string
    rtn += str.substr(0, position);
    // remove the back-ref pattern to continue parsing
    str = str.substr(position + static_cast<size_type>(m.length(0)));
  }
  if (!str.empty())  // add the remainder
    rtn += str;      // of the string
  return {rtn, backrefs};
}

}  // namespace

//
std::unique_ptr<column> replace_with_backrefs(
  strings_column_view const& strings,
  std::string const& pattern,
  std::string const& repl,
  rmm::cuda_stream_view stream,
  rmm::mr::device_memory_resource* mr = rmm::mr::get_current_device_resource())
{
  if (strings.is_empty()) return make_empty_column(type_id::STRING);

  CUDF_EXPECTS(!pattern.empty(), "Parameter pattern must not be empty");
  CUDF_EXPECTS(!repl.empty(), "Parameter repl must not be empty");

  auto d_strings = column_device_view::create(strings.parent(), stream);
  // compile regex into device object
  auto d_prog = reprog_device::create(pattern, get_character_flags_table(), strings.size(), stream);
  auto const regex_insts = d_prog->insts_counts();

  // parse the repl string for back-ref indicators
  auto const parse_result = parse_backrefs(repl);
  rmm::device_uvector<backref_type> backrefs(parse_result.second.size(), stream);
  CUDA_TRY(hipMemcpyAsync(backrefs.data(),
                           parse_result.second.data(),
                           sizeof(backref_type) * backrefs.size(),
                           hipMemcpyHostToDevice,
                           stream.value()));
  string_scalar repl_scalar(parse_result.first, true, stream);
  string_view const d_repl_template = repl_scalar.value();

  using BackRefIterator = decltype(backrefs.begin());

  // create child columns
  auto [offsets, chars] = [&] {
    if (regex_insts <= RX_SMALL_INSTS) {
      return make_strings_children(
        backrefs_fn<BackRefIterator, RX_STACK_SMALL>{
          *d_strings, *d_prog, d_repl_template, backrefs.begin(), backrefs.end()},
        strings.size(),
        stream,
        mr);
    } else if (regex_insts <= RX_MEDIUM_INSTS) {
      return make_strings_children(
        backrefs_fn<BackRefIterator, RX_STACK_MEDIUM>{
          *d_strings, *d_prog, d_repl_template, backrefs.begin(), backrefs.end()},
        strings.size(),
        stream,
        mr);
    } else if (regex_insts <= RX_LARGE_INSTS) {
      return make_strings_children(
        backrefs_fn<BackRefIterator, RX_STACK_LARGE>{
          *d_strings, *d_prog, d_repl_template, backrefs.begin(), backrefs.end()},
        strings.size(),
        stream,
        mr);
    } else {
      return make_strings_children(
        backrefs_fn<BackRefIterator, RX_STACK_ANY>{
          *d_strings, *d_prog, d_repl_template, backrefs.begin(), backrefs.end()},
        strings.size(),
        stream,
        mr);
    }
  }();

  return make_strings_column(strings.size(),
                             std::move(offsets),
                             std::move(chars),
                             strings.null_count(),
                             cudf::detail::copy_bitmask(strings.parent(), stream, mr));
}

}  // namespace detail

// external API

std::unique_ptr<column> replace_with_backrefs(strings_column_view const& strings,
                                              std::string const& pattern,
                                              std::string const& repl,
                                              rmm::mr::device_memory_resource* mr)
{
  CUDF_FUNC_RANGE();
  return detail::replace_with_backrefs(strings, pattern, repl, rmm::cuda_stream_default, mr);
}

}  // namespace strings
}  // namespace cudf
