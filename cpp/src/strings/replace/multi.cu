#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019-2024, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "strings/split/split.cuh"

#include <cudf/column/column_device_view.cuh>
#include <cudf/column/column_factories.hpp>
#include <cudf/detail/null_mask.hpp>
#include <cudf/detail/nvtx/ranges.hpp>
#include <cudf/detail/utilities/algorithm.cuh>
#include <cudf/detail/utilities/cuda.cuh>
#include <cudf/strings/detail/replace.hpp>
#include <cudf/strings/detail/strings_children.cuh>
#include <cudf/strings/detail/strings_column_factories.cuh>
#include <cudf/strings/detail/utilities.cuh>
#include <cudf/strings/replace.hpp>
#include <cudf/strings/string_view.cuh>
#include <cudf/strings/strings_column_view.hpp>
#include <cudf/utilities/default_stream.hpp>
#include <cudf/utilities/span.hpp>

#include <rmm/cuda_stream_view.hpp>
#include <rmm/device_uvector.hpp>
#include <rmm/resource_ref.hpp>

#include <cuda/functional>
#include <thrust/binary_search.h>
#include <thrust/copy.h>
#include <thrust/count.h>
#include <thrust/distance.h>
#include <thrust/execution_policy.h>
#include <thrust/for_each.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/iterator/zip_iterator.h>
#include <thrust/optional.h>
#include <thrust/scan.h>
#include <thrust/transform.h>

namespace cudf {
namespace strings {
namespace detail {
namespace {

/**
 * @brief Threshold to decide on using string or character-parallel functions.
 *
 * If the average byte length of a string in a column exceeds this value then
 * the character-parallel function is used.
 * Otherwise, a regular string-parallel function is used.
 *
 * This value was found using the replace-multi benchmark results using an
 * RTX A6000.
 */
constexpr size_type AVG_CHAR_BYTES_THRESHOLD = 256;

/**
 * @brief Type used for holding the target position (first) and the
 * target index (second).
 */
using target_pair = thrust::tuple<int64_t, size_type>;

/**
 * @brief Helper functions for performing character-parallel replace
 */
struct replace_multi_parallel_fn {
  __device__ char const* get_base_ptr() const { return d_strings.head<char>(); }

  __device__ string_view const get_string(size_type idx) const
  {
    return d_strings.element<string_view>(idx);
  }

  __device__ string_view const get_replacement_string(size_type idx) const
  {
    return d_replacements.size() == 1 ? d_replacements[0] : d_replacements[idx];
  }

  __device__ bool is_valid(size_type idx) const { return d_strings.is_valid(idx); }

  /**
   * @brief Returns the index of the target string found at the given byte position
   * in the input strings column
   *
   * @param idx Index of the byte position in the chars column
   * @param chars_bytes Number of bytes in the chars column
   */
  __device__ size_type target_index(int64_t idx, int64_t chars_bytes) const
  {
    auto const d_offsets = d_strings_offsets;
    auto const d_chars   = get_base_ptr() + d_offsets[0] + idx;
    size_type str_idx    = -1;
    string_view d_str{};
    for (std::size_t t = 0; t < d_targets.size(); ++t) {
      auto const d_tgt = d_targets[t];
      if (!d_tgt.empty() && (idx + d_tgt.size_bytes() <= chars_bytes) &&
          (d_tgt.compare(d_chars, d_tgt.size_bytes()) == 0)) {
        if (str_idx < 0) {
          auto const idx_itr =
            thrust::upper_bound(thrust::seq, d_offsets, d_offsets + d_strings.size(), idx);
          str_idx = thrust::distance(d_offsets, idx_itr) - 1;
          d_str   = get_string(str_idx - d_offsets[0]);
        }
        if ((d_chars + d_tgt.size_bytes()) <= (d_str.data() + d_str.size_bytes())) { return t; }
      }
    }
    return -1;
  }

  __device__ bool has_target(int64_t idx, int64_t chars_bytes) const
  {
    auto const d_chars = get_base_ptr() + d_strings_offsets[0] + idx;
    for (auto& d_tgt : d_targets) {
      if (!d_tgt.empty() && (idx + d_tgt.size_bytes() <= chars_bytes) &&
          (d_tgt.compare(d_chars, d_tgt.size_bytes()) == 0)) {
        return true;
      }
    }
    return false;
  }

  /**
   * @brief Count the number of strings that will be produced by the replace
   *
   * This includes segments of the string that are not replaced as well as those
   * that are replaced.
   *
   * @param idx Index of the row in d_strings to be processed
   * @param d_positions Positions of the targets found in the chars column
   * @param d_targets_offsets Offsets identify which target positions go with the current string
   * @return Number of substrings resulting from the replace operations on this row
   */
  __device__ size_type count_strings(size_type idx,
                                     int64_t const* d_positions,
                                     size_type const* d_indices,
                                     cudf::detail::input_offsetalator d_targets_offsets) const
  {
    if (!is_valid(idx)) { return 0; }

    auto const d_str     = get_string(idx);
    auto const d_str_end = d_str.data() + d_str.size_bytes();
    auto const base_ptr  = get_base_ptr();

    auto const target_offset = d_targets_offsets[idx];
    auto const targets_size  = static_cast<size_type>(d_targets_offsets[idx + 1] - target_offset);
    auto const positions     = d_positions + target_offset;
    auto const indices       = d_indices + target_offset;

    size_type count = 1;  // always at least one string
    auto str_ptr    = d_str.data();
    for (std::size_t i = 0; i < targets_size; ++i) {
      auto const tgt_idx = indices[i];
      auto const d_tgt   = d_targets[tgt_idx];
      auto const tgt_ptr = base_ptr + positions[i];
      if (str_ptr <= tgt_ptr && tgt_ptr < d_str_end) {
        auto const keep_size = static_cast<size_type>(thrust::distance(str_ptr, tgt_ptr));
        if (keep_size > 0) { count++; }  // don't bother counting empty strings

        auto const d_repl = get_replacement_string(tgt_idx);
        if (!d_repl.empty()) { count++; }

        str_ptr += keep_size + d_tgt.size_bytes();
      }
    }

    return count;
  }

  /**
   * @brief Retrieve the strings for each row
   *
   * This will return string segments as string_index_pair objects for
   * parts of the string that are not replaced interlaced with the
   * appropriate replacement string where replacement targets are found.
   *
   * This function is called only once to produce both the string_index_pair objects
   * and the output row size in bytes.
   *
   * @param idx Index of the row in d_strings
   * @param d_offsets Offsets to identify where to store the results of the replace for this string
   * @param d_positions The target positions found in the chars column
   * @param d_targets_offsets The offsets to identify which target positions go with this string
   * @param d_all_strings The output of all the produced string segments
   * @return The size in bytes of the output string for this row
   */
  __device__ size_type get_strings(size_type idx,
                                   cudf::detail::input_offsetalator const d_offsets,
                                   int64_t const* d_positions,
                                   size_type const* d_indices,
                                   cudf::detail::input_offsetalator d_targets_offsets,
                                   string_index_pair* d_all_strings) const
  {
    if (!is_valid(idx)) { return 0; }

    auto const d_output  = d_all_strings + d_offsets[idx];
    auto const d_str     = get_string(idx);
    auto const d_str_end = d_str.data() + d_str.size_bytes();
    auto const base_ptr  = get_base_ptr();

    auto const target_offset = d_targets_offsets[idx];
    auto const targets_size  = static_cast<size_type>(d_targets_offsets[idx + 1] - target_offset);
    auto const positions     = d_positions + target_offset;
    auto const indices       = d_indices + target_offset;

    size_type output_idx  = 0;
    size_type output_size = 0;
    auto str_ptr          = d_str.data();
    for (std::size_t i = 0; i < targets_size; ++i) {
      auto const tgt_idx = indices[i];
      auto const d_tgt   = d_targets[tgt_idx];
      auto const tgt_ptr = base_ptr + positions[i];
      if (str_ptr <= tgt_ptr && tgt_ptr < d_str_end) {
        auto const keep_size = static_cast<size_type>(thrust::distance(str_ptr, tgt_ptr));
        if (keep_size > 0) { d_output[output_idx++] = string_index_pair{str_ptr, keep_size}; }
        output_size += keep_size;

        auto const d_repl = get_replacement_string(tgt_idx);
        if (!d_repl.empty()) {
          d_output[output_idx++] = string_index_pair{d_repl.data(), d_repl.size_bytes()};
        }
        output_size += d_repl.size_bytes();

        str_ptr += keep_size + d_tgt.size_bytes();
      }
    }
    // include any leftover parts of the string
    if (str_ptr <= d_str_end) {
      auto const left_size = static_cast<size_type>(thrust::distance(str_ptr, d_str_end));
      d_output[output_idx] = string_index_pair{str_ptr, left_size};
      output_size += left_size;
    }
    return output_size;
  }

  replace_multi_parallel_fn(column_device_view const& d_strings,
                            cudf::detail::input_offsetalator d_strings_offsets,
                            device_span<string_view const> d_targets,
                            device_span<string_view const> d_replacements)
    : d_strings(d_strings),
      d_strings_offsets(d_strings_offsets),
      d_targets{d_targets},
      d_replacements{d_replacements}
  {
  }

 protected:
  column_device_view d_strings;
  cudf::detail::input_offsetalator d_strings_offsets;
  device_span<string_view const> d_targets;
  device_span<string_view const> d_replacements;
};

/**
 * @brief Used by the copy-if function to produce target_pair objects
 *
 * Using an inplace lambda caused a runtime crash in thrust::copy_if
 * (this happens sometimes when passing device lambdas to thrust algorithms)
 */
struct pair_generator {
  __device__ target_pair operator()(int64_t idx) const
  {
    return thrust::make_tuple(idx, fn.target_index(idx, chars_bytes));
  }
  replace_multi_parallel_fn fn;
  int64_t chars_bytes;
};

struct copy_if_fn {
  __device__ bool operator()(target_pair pos) { return thrust::get<1>(pos) >= 0; }
};

std::unique_ptr<column> replace_character_parallel(strings_column_view const& input,
                                                   strings_column_view const& targets,
                                                   strings_column_view const& repls,
                                                   rmm::cuda_stream_view stream,
                                                   rmm::device_async_resource_ref mr)
{
  auto d_strings = column_device_view::create(input.parent(), stream);

  auto const strings_count = input.size();
  auto const chars_bytes =
    get_offset_value(input.offsets(), input.offset() + strings_count, stream) -
    get_offset_value(input.offsets(), input.offset(), stream);

  auto d_targets =
    create_string_vector_from_column(targets, stream, rmm::mr::get_current_device_resource());
  auto d_replacements =
    create_string_vector_from_column(repls, stream, rmm::mr::get_current_device_resource());

  replace_multi_parallel_fn fn{
    *d_strings,
    cudf::detail::offsetalator_factory::make_input_iterator(input.offsets(), input.offset()),
    d_targets,
    d_replacements,
  };

  // Count the number of targets in the entire column.
  // Note this may over-count in the case where a target spans adjacent strings.
  auto target_count = thrust::count_if(
    rmm::exec_policy_nosync(stream),
    thrust::make_counting_iterator<int64_t>(0),
    thrust::make_counting_iterator<int64_t>(chars_bytes),
    [fn, chars_bytes] __device__(int64_t idx) { return fn.has_target(idx, chars_bytes); });

  // Create a vector of every target position in the chars column.
  // These may also include overlapping targets which will be resolved later.
  auto targets_positions = rmm::device_uvector<int64_t>(target_count, stream);
  auto targets_indices   = rmm::device_uvector<size_type>(target_count, stream);

  // cudf::detail::make_counting_transform_iterator hardcodes size_type
  auto const copy_itr = thrust::make_transform_iterator(thrust::counting_iterator<int64_t>(0),
                                                        pair_generator{fn, chars_bytes});
  auto const out_itr  = thrust::make_zip_iterator(
    thrust::make_tuple(targets_positions.begin(), targets_indices.begin()));
  auto const copy_end =
    cudf::detail::copy_if_safe(copy_itr, copy_itr + chars_bytes, out_itr, copy_if_fn{}, stream);

  // adjust target count since the copy-if may have eliminated some invalid targets
  target_count = std::min(static_cast<int64_t>(std::distance(out_itr, copy_end)), target_count);
  targets_positions.resize(target_count, stream);
  targets_indices.resize(target_count, stream);
  auto d_positions       = targets_positions.data();
  auto d_targets_indices = targets_indices.data();

  // create a vector of offsets to each string's set of target positions
  auto const targets_offsets = create_offsets_from_positions(
    input, targets_positions, stream, rmm::mr::get_current_device_resource());
  auto const d_targets_offsets =
    cudf::detail::offsetalator_factory::make_input_iterator(targets_offsets->view());

  // compute the number of string segments produced by replace in each string
  auto counts = rmm::device_uvector<size_type>(strings_count, stream);
  thrust::transform(rmm::exec_policy_nosync(stream),
                    thrust::counting_iterator<size_type>(0),
                    thrust::counting_iterator<size_type>(strings_count),
                    counts.begin(),
                    cuda::proclaim_return_type<size_type>(
                      [fn, d_positions, d_targets_indices, d_targets_offsets] __device__(
                        size_type idx) -> size_type {
                        return fn.count_strings(
                          idx, d_positions, d_targets_indices, d_targets_offsets);
                      }));

  // create offsets from the counts
  auto [offsets, total_strings] =
    cudf::detail::make_offsets_child_column(counts.begin(), counts.end(), stream, mr);
  auto const d_strings_offsets =
    cudf::detail::offsetalator_factory::make_input_iterator(offsets->view());

  // build a vector of all the positions for all the strings
  auto indices   = rmm::device_uvector<string_index_pair>(total_strings, stream);
  auto d_indices = indices.data();
  auto d_sizes   = counts.data();  // reusing this vector to hold output sizes now
  thrust::for_each_n(
    rmm::exec_policy_nosync(stream),
    thrust::make_counting_iterator<size_type>(0),
    strings_count,
    [fn,
     d_strings_offsets,
     d_positions,
     d_targets_indices,
     d_targets_offsets,
     d_indices,
     d_sizes] __device__(size_type idx) {
      d_sizes[idx] = fn.get_strings(
        idx, d_strings_offsets, d_positions, d_targets_indices, d_targets_offsets, d_indices);
    });

  // use this utility to gather the string parts into a contiguous chars column
  auto chars      = make_strings_column(indices.begin(), indices.end(), stream, mr);
  auto chars_data = chars->release().data;

  // create offsets from the sizes
  offsets = std::get<0>(
    cudf::strings::detail::make_offsets_child_column(counts.begin(), counts.end(), stream, mr));

  // build the strings columns from the chars and offsets
  return make_strings_column(strings_count,
                             std::move(offsets),
                             std::move(chars_data.release()[0]),
                             input.null_count(),
                             cudf::detail::copy_bitmask(input.parent(), stream, mr));
}

/**
 * @brief Function logic for the replace_string_parallel
 *
 * Performs the multi-replace operation with a thread per string.
 * This performs best on smaller strings. @see AVG_CHAR_BYTES_THRESHOLD
 */
struct replace_multi_fn {
  column_device_view const d_strings;
  column_device_view const d_targets;
  column_device_view const d_repls;
  size_type* d_sizes{};
  char* d_chars{};
  cudf::detail::input_offsetalator d_offsets;

  __device__ void operator()(size_type idx)
  {
    if (d_strings.is_null(idx)) {
      if (!d_chars) { d_sizes[idx] = 0; }
      return;
    }
    auto const d_str   = d_strings.element<string_view>(idx);
    char const* in_ptr = d_str.data();

    size_type bytes = d_str.size_bytes();
    size_type spos  = 0;
    size_type lpos  = 0;
    char* out_ptr   = d_chars ? d_chars + d_offsets[idx] : nullptr;

    // check each character against each target
    while (spos < d_str.size_bytes()) {
      for (int tgt_idx = 0; tgt_idx < d_targets.size(); ++tgt_idx) {
        auto const d_tgt = d_targets.element<string_view>(tgt_idx);
        if ((d_tgt.size_bytes() <= (d_str.size_bytes() - spos)) &&    // check fit
            (d_tgt.compare(in_ptr + spos, d_tgt.size_bytes()) == 0))  // and match
        {
          auto const d_repl = (d_repls.size() == 1) ? d_repls.element<string_view>(0)
                                                    : d_repls.element<string_view>(tgt_idx);
          bytes += d_repl.size_bytes() - d_tgt.size_bytes();
          if (out_ptr) {
            out_ptr = copy_and_increment(out_ptr, in_ptr + lpos, spos - lpos);
            out_ptr = copy_string(out_ptr, d_repl);
            lpos    = spos + d_tgt.size_bytes();
          }
          spos += d_tgt.size_bytes() - 1;
          break;
        }
      }
      ++spos;
    }
    if (out_ptr)  // copy remainder
    {
      memcpy(out_ptr, in_ptr + lpos, d_str.size_bytes() - lpos);
    } else {
      d_sizes[idx] = bytes;
    }
  }
};

std::unique_ptr<column> replace_string_parallel(strings_column_view const& input,
                                                strings_column_view const& targets,
                                                strings_column_view const& repls,
                                                rmm::cuda_stream_view stream,
                                                rmm::device_async_resource_ref mr)
{
  auto d_strings      = column_device_view::create(input.parent(), stream);
  auto d_targets      = column_device_view::create(targets.parent(), stream);
  auto d_replacements = column_device_view::create(repls.parent(), stream);

  auto [offsets_column, chars] = make_strings_children(
    replace_multi_fn{*d_strings, *d_targets, *d_replacements}, input.size(), stream, mr);

  return make_strings_column(input.size(),
                             std::move(offsets_column),
                             chars.release(),
                             input.null_count(),
                             cudf::detail::copy_bitmask(input.parent(), stream, mr));
}

}  // namespace

std::unique_ptr<column> replace(strings_column_view const& input,
                                strings_column_view const& targets,
                                strings_column_view const& repls,
                                rmm::cuda_stream_view stream,
                                rmm::device_async_resource_ref mr)
{
  if (input.is_empty()) { return make_empty_column(type_id::STRING); }
  CUDF_EXPECTS(((targets.size() > 0) && (targets.null_count() == 0)),
               "Parameters targets must not be empty and must not have nulls");
  CUDF_EXPECTS(((repls.size() > 0) && (repls.null_count() == 0)),
               "Parameters repls must not be empty and must not have nulls");
  if (repls.size() > 1)
    CUDF_EXPECTS(repls.size() == targets.size(), "Sizes for targets and repls must match");

  return (input.size() == input.null_count() ||
          ((input.chars_size(stream) / (input.size() - input.null_count())) <
           AVG_CHAR_BYTES_THRESHOLD))
           ? replace_string_parallel(input, targets, repls, stream, mr)
           : replace_character_parallel(input, targets, repls, stream, mr);
}

}  // namespace detail

// external API

std::unique_ptr<column> replace(strings_column_view const& strings,
                                strings_column_view const& targets,
                                strings_column_view const& repls,
                                rmm::cuda_stream_view stream,
                                rmm::device_async_resource_ref mr)
{
  CUDF_FUNC_RANGE();
  return detail::replace(strings, targets, repls, stream, mr);
}

}  // namespace strings
}  // namespace cudf
