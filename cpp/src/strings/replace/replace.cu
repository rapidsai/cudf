#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019-2024, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "strings/split/split.cuh"

#include <cudf/column/column_device_view.cuh>
#include <cudf/column/column_factories.hpp>
#include <cudf/detail/null_mask.hpp>
#include <cudf/detail/nvtx/ranges.hpp>
#include <cudf/detail/utilities/algorithm.cuh>
#include <cudf/strings/detail/replace.hpp>
#include <cudf/strings/detail/strings_children.cuh>
#include <cudf/strings/detail/strings_column_factories.cuh>
#include <cudf/strings/detail/utilities.cuh>
#include <cudf/strings/replace.hpp>
#include <cudf/strings/string_view.cuh>
#include <cudf/strings/strings_column_view.hpp>
#include <cudf/utilities/default_stream.hpp>

#include <rmm/cuda_stream_view.hpp>
#include <rmm/device_uvector.hpp>
#include <rmm/resource_ref.hpp>

#include <cuda/functional>
#include <thrust/binary_search.h>
#include <thrust/copy.h>
#include <thrust/count.h>
#include <thrust/distance.h>
#include <thrust/execution_policy.h>
#include <thrust/for_each.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/transform.h>

namespace cudf {
namespace strings {
namespace detail {
namespace {

/**
 * @brief Threshold to decide on using string or character-parallel functions.
 *
 * If the average byte length of a string in a column exceeds this value then
 * the character-parallel function is used.
 * Otherwise, a regular string-parallel function is used.
 *
 * This value was found using the replace-multi benchmark results using an
 * RTX A6000.
 */
constexpr size_type AVG_CHAR_BYTES_THRESHOLD = 256;

/**
 * @brief Helper functions for performing character-parallel replace
 */
struct replace_parallel_chars_fn {
  __device__ inline char const* get_base_ptr() const { return d_strings.head<char>(); }

  __device__ inline string_view const get_string(size_type idx) const
  {
    return d_strings.element<string_view>(idx);
  }

  __device__ inline bool is_valid(size_type idx) const { return d_strings.is_valid(idx); }

  /**
   * @brief Returns true if the target string is found at the given byte position
   * in the input strings column and is legally within a string row
   *
   * @param idx Index of the byte position in the chars column
   */
  __device__ bool is_target_within_row(int64_t idx) const
  {
    auto const d_offsets = d_strings_offsets;
    auto const d_chars   = get_base_ptr() + idx;
    auto const d_tgt     = d_target;
    auto const chars_end = chars_bytes + d_offsets[0];
    if (!d_tgt.empty() && (idx + d_tgt.size_bytes() <= chars_end) &&
        (d_tgt.compare(d_chars, d_tgt.size_bytes()) == 0)) {
      auto const idx_itr =
        thrust::upper_bound(thrust::seq, d_offsets, d_offsets + d_strings.size(), idx);
      auto str_idx = static_cast<size_type>(thrust::distance(d_offsets, idx_itr) - 1);
      auto d_str   = get_string(str_idx);
      if ((d_chars + d_tgt.size_bytes()) <= (d_str.data() + d_str.size_bytes())) { return true; }
    }
    return false;
  }

  /**
   * @brief Returns true if the target string found at the given byte position
   *
   * @param idx Index of the byte position in the chars column
   */
  __device__ bool has_target(int64_t idx) const
  {
    auto const d_chars = get_base_ptr() + d_strings_offsets[0] + idx;
    return (!d_target.empty() && (idx + d_target.size_bytes() <= chars_bytes) &&
            (d_target.compare(d_chars, d_target.size_bytes()) == 0));
  }

  /**
   * @brief Count the number of strings that will be produced by the replace
   *
   * This includes segments of the string that are not replaced as well as those
   * that are replaced.
   *
   * @param idx Index of the row in d_strings to be processed
   * @param d_positions Positions of the targets found in the chars column
   * @param d_targets_offsets Offsets identify which target positions go with the current string
   * @return Number of substrings resulting from the replace operations on this row
   */
  __device__ size_type count_strings(size_type idx,
                                     int64_t const* d_positions,
                                     cudf::detail::input_offsetalator d_targets_offsets) const
  {
    if (!is_valid(idx)) { return 0; }

    auto const d_str     = get_string(idx);
    auto const d_str_end = d_str.data() + d_str.size_bytes();
    auto const base_ptr  = get_base_ptr();
    auto max_n           = (maxrepl < 0) ? d_str.length() : maxrepl;

    auto const target_offset = d_targets_offsets[idx];
    auto const targets_size  = static_cast<size_type>(d_targets_offsets[idx + 1] - target_offset);
    auto const positions     = d_positions + target_offset;

    size_type count = 1;  // always at least one string
    auto str_ptr    = d_str.data();
    for (std::size_t i = 0; (i < targets_size) && (max_n > 0); ++i) {
      auto const tgt_ptr = base_ptr + positions[i];
      if (str_ptr <= tgt_ptr && tgt_ptr < d_str_end) {
        auto const keep_size = static_cast<size_type>(thrust::distance(str_ptr, tgt_ptr));
        if (keep_size > 0) { count++; }  // don't bother counting empty strings
        if (!d_replacement.empty()) { count++; }
        str_ptr += keep_size + d_target.size_bytes();
        --max_n;
      }
    }
    return count;
  }

  /**
   * @brief Retrieve the strings for each row
   *
   * This will return string segments as string_index_pair objects for
   * parts of the string that are not replaced interlaced with the
   * appropriate replacement string where replacement targets are found.
   *
   * This function is called only once to produce both the string_index_pair objects
   * and the output row size in bytes.
   *
   * @param idx Index of the row in d_strings
   * @param d_offsets Offsets to identify where to store the results of the replace for this string
   * @param d_positions The target positions found in the chars column
   * @param d_targets_offsets The offsets to identify which target positions go with this string
   * @param d_all_strings The output of all the produced string segments
   * @return The size in bytes of the output string for this row
   */
  __device__ size_type get_strings(size_type idx,
                                   cudf::detail::input_offsetalator const d_offsets,
                                   int64_t const* d_positions,
                                   cudf::detail::input_offsetalator d_targets_offsets,
                                   string_index_pair* d_all_strings) const
  {
    if (!is_valid(idx)) { return 0; }

    auto const d_output  = d_all_strings + d_offsets[idx];
    auto const d_str     = get_string(idx);
    auto const d_str_end = d_str.data() + d_str.size_bytes();
    auto const base_ptr  = get_base_ptr();
    auto max_n           = (maxrepl < 0) ? d_str.length() : maxrepl;

    auto const target_offset = d_targets_offsets[idx];
    auto const targets_size  = static_cast<size_type>(d_targets_offsets[idx + 1] - target_offset);
    auto const positions     = d_positions + target_offset;

    size_type output_idx  = 0;
    size_type output_size = 0;
    auto str_ptr          = d_str.data();
    for (std::size_t i = 0; (i < targets_size) && (max_n > 0); ++i) {
      auto const tgt_ptr = base_ptr + positions[i];
      if (str_ptr <= tgt_ptr && tgt_ptr < d_str_end) {
        auto const keep_size = static_cast<size_type>(thrust::distance(str_ptr, tgt_ptr));
        if (keep_size > 0) { d_output[output_idx++] = string_index_pair{str_ptr, keep_size}; }
        output_size += keep_size;

        if (!d_replacement.empty()) {
          d_output[output_idx++] =
            string_index_pair{d_replacement.data(), d_replacement.size_bytes()};
        }
        output_size += d_replacement.size_bytes();

        str_ptr += keep_size + d_target.size_bytes();
        --max_n;
      }
    }
    // include any leftover parts of the string
    if (str_ptr <= d_str_end) {
      auto const left_size = static_cast<size_type>(thrust::distance(str_ptr, d_str_end));
      d_output[output_idx] = string_index_pair{str_ptr, left_size};
      output_size += left_size;
    }
    return output_size;
  }

  replace_parallel_chars_fn(column_device_view const& d_strings,
                            cudf::detail::input_offsetalator d_strings_offsets,
                            int64_t chars_bytes,
                            string_view d_target,
                            string_view d_replacement,
                            cudf::size_type maxrepl)
    : d_strings(d_strings),
      d_strings_offsets(d_strings_offsets),
      chars_bytes(chars_bytes),
      d_target{d_target},
      d_replacement{d_replacement},
      maxrepl(maxrepl)
  {
  }

 protected:
  column_device_view d_strings;
  cudf::detail::input_offsetalator d_strings_offsets;
  int64_t chars_bytes;
  string_view d_target;
  string_view d_replacement;
  cudf::size_type maxrepl;
};

std::unique_ptr<column> replace_character_parallel(strings_column_view const& input,
                                                   string_view const& d_target,
                                                   string_view const& d_replacement,
                                                   cudf::size_type maxrepl,
                                                   rmm::cuda_stream_view stream,
                                                   rmm::device_async_resource_ref mr)
{
  auto d_strings = column_device_view::create(input.parent(), stream);

  auto const strings_count = input.size();
  auto const chars_offset  = get_offset_value(input.offsets(), input.offset(), stream);
  auto const chars_bytes =
    get_offset_value(input.offsets(), input.offset() + strings_count, stream) - chars_offset;

  auto const offsets_begin =
    cudf::detail::offsetalator_factory::make_input_iterator(input.offsets(), input.offset());

  replace_parallel_chars_fn fn{
    *d_strings, offsets_begin, chars_bytes, d_target, d_replacement, maxrepl};

  // Count the number of targets in the entire column.
  // Note this may over-count in the case where a target spans adjacent strings.
  auto target_count = thrust::count_if(rmm::exec_policy_nosync(stream),
                                       thrust::make_counting_iterator<int64_t>(0),
                                       thrust::make_counting_iterator<int64_t>(chars_bytes),
                                       [fn] __device__(int64_t idx) { return fn.has_target(idx); });

  // Create a vector of every target position in the chars column.
  // These may also include overlapping targets which will be resolved later.
  auto targets_positions = rmm::device_uvector<int64_t>(target_count, stream);
  auto const copy_itr    = thrust::counting_iterator<int64_t>(chars_offset);
  auto const copy_end    = cudf::detail::copy_if_safe(
    copy_itr,
    copy_itr + chars_bytes + chars_offset,
    targets_positions.begin(),
    [fn] __device__(int64_t idx) { return fn.is_target_within_row(idx); },
    stream);

  // adjust target count since the copy-if may have eliminated some invalid targets
  target_count = std::min(std::distance(targets_positions.begin(), copy_end), target_count);
  targets_positions.resize(target_count, stream);
  auto d_positions = targets_positions.data();

  // create a vector of offsets to each string's set of target positions
  auto const targets_offsets = create_offsets_from_positions(
    input, targets_positions, stream, rmm::mr::get_current_device_resource());
  auto const d_targets_offsets =
    cudf::detail::offsetalator_factory::make_input_iterator(targets_offsets->view());

  // compute the number of string segments produced by replace in each string
  auto counts = rmm::device_uvector<size_type>(strings_count, stream);
  thrust::transform(rmm::exec_policy_nosync(stream),
                    thrust::counting_iterator<size_type>(0),
                    thrust::counting_iterator<size_type>(strings_count),
                    counts.begin(),
                    cuda::proclaim_return_type<size_type>(
                      [fn, d_positions, d_targets_offsets] __device__(size_type idx) -> size_type {
                        return fn.count_strings(idx, d_positions, d_targets_offsets);
                      }));

  // create offsets from the counts
  auto [offsets, total_strings] =
    cudf::detail::make_offsets_child_column(counts.begin(), counts.end(), stream, mr);
  auto const d_strings_offsets =
    cudf::detail::offsetalator_factory::make_input_iterator(offsets->view());

  // build a vector of all the positions for all the strings
  auto indices   = rmm::device_uvector<string_index_pair>(total_strings, stream);
  auto d_indices = indices.data();
  auto d_sizes   = counts.data();  // reusing this vector to hold output sizes now
  thrust::for_each_n(
    rmm::exec_policy_nosync(stream),
    thrust::make_counting_iterator<size_type>(0),
    strings_count,
    [fn, d_strings_offsets, d_positions, d_targets_offsets, d_indices, d_sizes] __device__(
      size_type idx) {
      d_sizes[idx] =
        fn.get_strings(idx, d_strings_offsets, d_positions, d_targets_offsets, d_indices);
    });

  // use this utility to gather the string parts into a contiguous chars column
  auto chars      = make_strings_column(indices.begin(), indices.end(), stream, mr);
  auto chars_data = chars->release().data;

  // create offsets from the sizes
  offsets = std::get<0>(
    cudf::strings::detail::make_offsets_child_column(counts.begin(), counts.end(), stream, mr));

  // build the strings columns from the chars and offsets
  return make_strings_column(strings_count,
                             std::move(offsets),
                             std::move(chars_data.release()[0]),
                             input.null_count(),
                             cudf::detail::copy_bitmask(input.parent(), stream, mr));
}

/**
 * @brief Function logic for the replace_string_parallel
 *
 * Performs the multi-replace operation with a thread per string.
 * This performs best on smaller strings. @see AVG_CHAR_BYTES_THRESHOLD
 */
struct replace_fn {
  column_device_view const d_strings;
  string_view d_target;
  string_view d_replacement;
  cudf::size_type maxrepl;
  cudf::size_type* d_sizes{};
  char* d_chars{};
  cudf::detail::input_offsetalator d_offsets;

  __device__ void operator()(size_type idx)
  {
    if (d_strings.is_null(idx)) {
      if (!d_chars) { d_sizes[idx] = 0; }
      return;
    }
    auto const d_str   = d_strings.element<string_view>(idx);
    char const* in_ptr = d_str.data();

    size_type bytes = d_str.size_bytes();
    size_type spos  = 0;
    size_type lpos  = 0;
    char* out_ptr   = d_chars ? d_chars + d_offsets[idx] : nullptr;
    auto max_n      = (maxrepl < 0) ? d_str.length() : maxrepl;

    // check each character against each target
    while (spos < d_str.size_bytes() && (max_n > 0)) {
      auto const d_tgt = d_target;
      if ((d_tgt.size_bytes() <= (d_str.size_bytes() - spos)) &&    // check fit
          (d_tgt.compare(in_ptr + spos, d_tgt.size_bytes()) == 0))  // and match
      {
        auto const d_repl = d_replacement;
        bytes += d_repl.size_bytes() - d_tgt.size_bytes();
        if (out_ptr) {
          out_ptr = copy_and_increment(out_ptr, in_ptr + lpos, spos - lpos);
          out_ptr = copy_string(out_ptr, d_repl);
          lpos    = spos + d_tgt.size_bytes();
        }
        spos += d_tgt.size_bytes() - 1;
        --max_n;
      }
      ++spos;
    }
    if (out_ptr) {  // copy remainder
      memcpy(out_ptr, in_ptr + lpos, d_str.size_bytes() - lpos);
    } else {
      d_sizes[idx] = bytes;
    }
  }
};

std::unique_ptr<column> replace_string_parallel(strings_column_view const& input,
                                                string_view const& d_target,
                                                string_view const& d_replacement,
                                                cudf::size_type maxrepl,
                                                rmm::cuda_stream_view stream,
                                                rmm::device_async_resource_ref mr)
{
  auto d_strings = column_device_view::create(input.parent(), stream);

  auto [offsets_column, chars] = make_strings_children(
    replace_fn{*d_strings, d_target, d_replacement, maxrepl}, input.size(), stream, mr);

  return make_strings_column(input.size(),
                             std::move(offsets_column),
                             chars.release(),
                             input.null_count(),
                             cudf::detail::copy_bitmask(input.parent(), stream, mr));
}

}  // namespace

std::unique_ptr<column> replace(strings_column_view const& input,
                                string_scalar const& target,
                                string_scalar const& repl,
                                cudf::size_type maxrepl,
                                rmm::cuda_stream_view stream,
                                rmm::device_async_resource_ref mr)
{
  if (input.is_empty()) { return make_empty_column(type_id::STRING); }
  if (maxrepl == 0) { return std::make_unique<cudf::column>(input.parent(), stream, mr); }
  CUDF_EXPECTS(repl.is_valid(stream), "Parameter repl must be valid.");
  CUDF_EXPECTS(target.is_valid(stream), "Parameter target must be valid.");
  CUDF_EXPECTS(target.size() > 0, "Parameter target must not be empty string.");

  string_view d_target(target.data(), target.size());
  string_view d_repl(repl.data(), repl.size());

  return (input.size() == input.null_count() ||
          ((input.chars_size(stream) / (input.size() - input.null_count())) <
           AVG_CHAR_BYTES_THRESHOLD))
           ? replace_string_parallel(input, d_target, d_repl, maxrepl, stream, mr)
           : replace_character_parallel(input, d_target, d_repl, maxrepl, stream, mr);
}

}  // namespace detail

// external API

std::unique_ptr<column> replace(strings_column_view const& strings,
                                string_scalar const& target,
                                string_scalar const& repl,
                                cudf::size_type maxrepl,
                                rmm::cuda_stream_view stream,
                                rmm::device_async_resource_ref mr)
{
  CUDF_FUNC_RANGE();
  return detail::replace(strings, target, repl, maxrepl, stream, mr);
}

}  // namespace strings
}  // namespace cudf
