/*
 * Copyright (c) 2020, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "backref_re.cuh"

#include <cudf/strings/detail/utilities.hpp>

#include <rmm/cuda_stream_view.hpp>

namespace cudf {
namespace strings {
namespace detail {

//
children_pair replace_with_backrefs_medium(column_device_view const& d_strings,
                                           reprog_device& d_prog,
                                           string_view const& d_repl_template,
                                           rmm::device_vector<backref_type>& backrefs,
                                           size_type null_count,
                                           rmm::cuda_stream_view stream,
                                           rmm::mr::device_memory_resource* mr)
{
  return make_strings_children(
    backrefs_fn<RX_STACK_MEDIUM>{
      d_strings, d_prog, d_repl_template, backrefs.begin(), backrefs.end()},
    d_strings.size(),
    null_count,
    stream,
    mr);
}

}  // namespace detail
}  // namespace strings
}  // namespace cudf
