#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2021-2024, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <cudf/column/column_device_view.cuh>
#include <cudf/column/column_factories.hpp>
#include <cudf/detail/copy.hpp>
#include <cudf/detail/get_value.cuh>
#include <cudf/detail/iterator.cuh>
#include <cudf/strings/detail/copying.hpp>
#include <cudf/strings/detail/strings_children.cuh>
#include <cudf/strings/detail/utilities.hpp>
#include <cudf/utilities/memory_resource.hpp>

#include <rmm/cuda_stream_view.hpp>
#include <rmm/exec_policy.hpp>

#include <thrust/iterator/counting_iterator.h>
#include <thrust/transform.h>

namespace cudf::strings::detail {

namespace {

struct output_sizes_fn {
  column_device_view const d_column;  // input strings column
  string_view const d_filler;
  size_type const offset;

  __device__ size_type get_string_size_at(size_type idx)
  {
    return d_column.is_null(idx) ? 0 : d_column.element<string_view>(idx).size_bytes();
  }

  __device__ size_type operator()(size_type idx)
  {
    auto const last_index = offset < 0 ? d_column.size() + offset : offset;
    if (offset < 0) {
      // shift left:  a,b,c,d,e,f -> b,c,d,e,f,x
      return (idx < last_index) ? get_string_size_at(idx - offset) : d_filler.size_bytes();
    } else {
      // shift right:  a,b,c,d,e,f -> x,a,b,c,d,e
      return (idx < last_index) ? d_filler.size_bytes() : get_string_size_at(idx - offset);
    }
  }
};

struct shift_chars_fn {
  column_device_view const d_column;  // input strings column
  string_view const d_filler;
  int64_t const offset;

  __device__ char operator()(int64_t idx)
  {
    if (offset < 0) {
      auto const last_index = -offset;
      if (idx < last_index) {
        auto const offsets     = d_column.child(strings_column_view::offsets_column_index);
        auto const off_itr     = cudf::detail::input_offsetalator(offsets.head(), offsets.type());
        auto const first_index = offset + off_itr[d_column.offset() + d_column.size()];
        return d_column.head<char>()[idx + first_index];
      } else {
        auto const char_index = idx - last_index;
        return d_filler.data()[char_index % d_filler.size_bytes()];
      }
    } else {
      if (idx < offset) {
        return d_filler.data()[idx % d_filler.size_bytes()];
      } else {
        auto const offsets = d_column.child(strings_column_view::offsets_column_index);
        auto const off_itr = cudf::detail::input_offsetalator(offsets.head(), offsets.type());
        return d_column.head<char>()[idx - offset + off_itr[d_column.offset()]];
      }
    }
  }
};

}  // namespace

std::unique_ptr<column> shift(strings_column_view const& input,
                              size_type offset,
                              scalar const& fill_value,
                              rmm::cuda_stream_view stream,
                              rmm::device_async_resource_ref mr)
{
  auto d_fill_str = static_cast<string_scalar const&>(fill_value).value(stream);

  // adjust offset when greater than the size of the input
  if (std::abs(offset) > input.size()) { offset = input.size(); }

  // build the output offsets by computing the sizes of each output row
  auto const d_input = column_device_view::create(input.parent(), stream);
  auto sizes_itr     = cudf::detail::make_counting_transform_iterator(
    0, output_sizes_fn{*d_input, d_fill_str, offset});
  auto [offsets_column, total_bytes] = cudf::strings::detail::make_offsets_child_column(
    sizes_itr, sizes_itr + input.size(), stream, mr);
  auto offsets_view = offsets_column->view();

  // compute the shift-offset for the output characters child column
  auto const shift_offset = [&] {
    auto const index = (offset < 0) ? input.size() + offset : offset;
    return (offset < 0 ? -1 : 1) * get_offset_value(offsets_view, index, stream);
  }();

  // create output chars child column
  rmm::device_uvector<char> chars(total_bytes, stream, mr);
  auto d_chars = chars.data();

  // run kernel to shift all the characters
  thrust::transform(rmm::exec_policy(stream),
                    thrust::counting_iterator<int64_t>(0),
                    thrust::counting_iterator<int64_t>(total_bytes),
                    d_chars,
                    shift_chars_fn{*d_input, d_fill_str, shift_offset});

  // caller sets the null-mask
  return make_strings_column(
    input.size(), std::move(offsets_column), chars.release(), 0, rmm::device_buffer{});
}

}  // namespace cudf::strings::detail
