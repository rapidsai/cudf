/*
 * Copyright (c) 2019-2020, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <cudf/column/column_device_view.cuh>
#include <cudf/column/column_factories.hpp>
#include <cudf/detail/gather.hpp>
#include <cudf/detail/utilities/integer_utils.hpp>
#include <cudf/strings/copying.hpp>
#include <cudf/strings/detail/utilities.hpp>
#include <cudf/strings/strings_column_view.hpp>

#include <thrust/sequence.h>

namespace cudf {
namespace strings {
namespace detail {
// new strings column from subset of this strings instance
std::unique_ptr<cudf::column> copy_slice(strings_column_view const& strings,
                                         size_type start,
                                         size_type end,
                                         size_type step,
                                         hipStream_t stream,
                                         rmm::mr::device_memory_resource* mr)
{
  size_type strings_count = strings.size();
  if (strings_count == 0) return make_empty_strings_column(mr, stream);
  if (step == 0) step = 1;
  CUDF_EXPECTS(step > 0, "Parameter step must be positive integer.");
  if (end < 0 || end > strings_count) end = strings_count;
  CUDF_EXPECTS(((start >= 0) && (start < end)), "Invalid start parameter value.");
  strings_count = cudf::util::round_up_safe<size_type>((end - start), step);
  //
  auto execpol = rmm::exec_policy(stream);
  // build indices
  rmm::device_vector<size_type> indices(strings_count);
  thrust::sequence(execpol->on(stream), indices.begin(), indices.end(), start, step);
  // create a column_view as a wrapper of these indices
  column_view indices_view(
    data_type{type_id::INT32}, strings_count, indices.data().get(), nullptr, 0);
  // build a new strings column from the indices
  auto sliced_table = cudf::detail::gather(table_view{{strings.parent()}},
                                           indices_view,
                                           cudf::detail::out_of_bounds_policy::NULLIFY,
                                           cudf::detail::negative_index_policy::NOT_ALLOWED,
                                           mr,
                                           stream)
                        ->release();
  std::unique_ptr<column> output_column(std::move(sliced_table.front()));
  if (output_column->null_count() == 0)
    output_column->set_null_mask(rmm::device_buffer{0, stream, mr}, 0);
  return output_column;
}

}  // namespace detail
}  // namespace strings
}  // namespace cudf
