#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019-2020, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <cudf/column/column_device_view.cuh>
#include <cudf/column/column_factories.hpp>
#include <cudf/copying.hpp>
#include <cudf/detail/concatenate.cuh>
#include <cudf/detail/null_mask.hpp>
#include <cudf/detail/utilities/cuda.cuh>
#include <cudf/strings/detail/concatenate.hpp>
#include <cudf/strings/detail/utilities.hpp>
#include <cudf/strings/strings_column_view.hpp>

#include <rmm/cuda_stream_view.hpp>

#include <thrust/binary_search.h>
#include <thrust/for_each.h>
#include <thrust/transform_reduce.h>
#include <thrust/transform_scan.h>

namespace cudf {
namespace strings {
namespace detail {
// Benchmark data, shared at https://github.com/rapidsai/cudf/pull/4703, shows
// that the single kernel optimization generally performs better, but when the
// number of chars/col is beyond a certain threshold memcpy performs better.
// This heuristic estimates which strategy will give better performance by
// comparing the mean chars/col with values from the above table.
constexpr bool use_fused_kernel_heuristic(bool const has_nulls,
                                          size_t const total_bytes,
                                          size_t const num_columns)
{
  return has_nulls ? total_bytes < num_columns * 1572864  // midpoint of 1048576 and 2097152
                   : total_bytes < num_columns * 393216;  // midpoint of 262144 and 524288
}

// Using a functor instead of a lambda as a workaround for:
// error: The enclosing parent function ("create_strings_device_views") for an
// extended __device__ lambda must not have deduced return type
struct chars_size_transform {
  __device__ size_t operator()(column_device_view const& col) const
  {
    if (col.size() > 0) {
      constexpr auto offsets_index = strings_column_view::offsets_column_index;
      auto d_offsets               = col.child(offsets_index).data<int32_t>();
      return d_offsets[col.size() + col.offset()] - d_offsets[col.offset()];
    } else {
      return 0;
    }
  }
};

auto create_strings_device_views(std::vector<column_view> const& views,
                                 rmm::cuda_stream_view stream)
{
  // Create device views for each input view
  using CDViewPtr = decltype(
    column_device_view::create(std::declval<column_view>(), std::declval<rmm::cuda_stream_view>()));
  auto device_view_owners = std::vector<CDViewPtr>(views.size());
  std::transform(
    views.cbegin(), views.cend(), device_view_owners.begin(), [stream](auto const& col) {
      return column_device_view::create(col, stream);
    });

  // Assemble contiguous array of device views
  auto device_views = thrust::host_vector<column_device_view>();
  device_views.reserve(views.size());
  std::transform(device_view_owners.cbegin(),
                 device_view_owners.cend(),
                 std::back_inserter(device_views),
                 [](auto const& col) { return *col; });
  auto d_views = rmm::device_vector<column_device_view>{device_views};

  // Compute the partition offsets and size of offset column
  // Note: Using 64-bit size_t so we can detect overflow of 32-bit size_type
  auto input_offsets = thrust::host_vector<size_t>(views.size() + 1);
  thrust::transform_inclusive_scan(
    thrust::host,
    device_views.cbegin(),
    device_views.cend(),
    std::next(input_offsets.begin()),
    [](auto const& col) { return static_cast<size_t>(col.size()); },
    thrust::plus<size_t>{});
  auto const d_input_offsets = rmm::device_vector<size_t>{input_offsets};
  auto const output_size     = input_offsets.back();

  // Compute the partition offsets and size of chars column
  // Note: Using 64-bit size_t so we can detect overflow of 32-bit size_type
  // Note: Using separate transform and inclusive_scan because
  // transform_inclusive_scan fails to compile with:
  // error: the default constructor of "cudf::column_device_view" cannot be
  // referenced -- it is a deleted function
  auto d_partition_offsets = rmm::device_vector<size_t>(views.size() + 1);
  thrust::transform(rmm::exec_policy(stream)->on(stream.value()),
                    d_views.cbegin(),
                    d_views.cend(),
                    std::next(d_partition_offsets.begin()),
                    chars_size_transform{});
  thrust::inclusive_scan(rmm::exec_policy(stream)->on(stream.value()),
                         d_partition_offsets.cbegin(),
                         d_partition_offsets.cend(),
                         d_partition_offsets.begin());
  auto const output_chars_size = d_partition_offsets.back();

  return std::make_tuple(std::move(device_view_owners),
                         std::move(d_views),
                         std::move(d_input_offsets),
                         std::move(d_partition_offsets),
                         output_size,
                         output_chars_size);
}

template <size_type block_size, bool Nullable>
__global__ void fused_concatenate_string_offset_kernel(column_device_view const* input_views,
                                                       size_t const* input_offsets,
                                                       size_t const* partition_offsets,
                                                       size_type const num_input_views,
                                                       size_type const output_size,
                                                       size_type* output_data,
                                                       bitmask_type* output_mask,
                                                       size_type* out_valid_count)
{
  size_type output_index     = threadIdx.x + blockIdx.x * blockDim.x;
  size_type warp_valid_count = 0;

  unsigned active_mask;
  if (Nullable) { active_mask = __ballot_sync(0xFFFF'FFFF, output_index < output_size); }
  while (output_index < output_size) {
    // Lookup input index by searching for output index in offsets
    // thrust::prev isn't in CUDA 10.0, so subtracting 1 here instead
    auto const offset_it =
      -1 + thrust::upper_bound(
             thrust::seq, input_offsets, input_offsets + num_input_views, output_index);
    size_type const partition_index = offset_it - input_offsets;

    auto const offset_index      = output_index - *offset_it;
    auto const& input_view       = input_views[partition_index];
    constexpr auto offsets_child = strings_column_view::offsets_column_index;
    auto const* input_data       = input_view.child(offsets_child).data<int32_t>();
    output_data[output_index] =
      input_data[offset_index + input_view.offset()]  // handle parent offset
      - input_data[input_view.offset()]               // subract first offset if non-zero
      + partition_offsets[partition_index];           // add offset of source column

    if (Nullable) {
      bool const bit_is_set       = input_view.is_valid(offset_index);
      bitmask_type const new_word = __ballot_sync(active_mask, bit_is_set);

      // First thread writes bitmask word
      if (threadIdx.x % cudf::detail::warp_size == 0) {
        output_mask[word_index(output_index)] = new_word;
      }

      warp_valid_count += __popc(new_word);
    }

    output_index += blockDim.x * gridDim.x;
    if (Nullable) { active_mask = __ballot_sync(active_mask, output_index < output_size); }
  }

  // Fill final offsets index with total size of char data
  if (output_index == output_size) {
    output_data[output_size] = partition_offsets[num_input_views];
  }

  if (Nullable) {
    using cudf::detail::single_lane_block_sum_reduce;
    auto block_valid_count = single_lane_block_sum_reduce<block_size, 0>(warp_valid_count);
    if (threadIdx.x == 0) { atomicAdd(out_valid_count, block_valid_count); }
  }
}

__global__ void fused_concatenate_string_chars_kernel(column_device_view const* input_views,
                                                      size_t const* partition_offsets,
                                                      size_type const num_input_views,
                                                      size_type const output_size,
                                                      char* output_data)
{
  size_type output_index = threadIdx.x + blockIdx.x * blockDim.x;

  while (output_index < output_size) {
    // Lookup input index by searching for output index in offsets
    // thrust::prev isn't in CUDA 10.0, so subtracting 1 here instead
    auto const offset_it =
      -1 + thrust::upper_bound(
             thrust::seq, partition_offsets, partition_offsets + num_input_views, output_index);
    size_type const partition_index = offset_it - partition_offsets;

    auto const offset_index = output_index - *offset_it;
    auto const& input_view  = input_views[partition_index];

    constexpr auto offsets_child   = strings_column_view::offsets_column_index;
    auto const* input_offsets_data = input_view.child(offsets_child).data<int32_t>();

    constexpr auto chars_child   = strings_column_view::chars_column_index;
    auto const* input_chars_data = input_view.child(chars_child).data<char>();

    auto const first_char     = input_offsets_data[input_view.offset()];
    output_data[output_index] = input_chars_data[offset_index + first_char];

    output_index += blockDim.x * gridDim.x;
  }
}

std::unique_ptr<column> concatenate(std::vector<column_view> const& columns,
                                    rmm::cuda_stream_view stream,
                                    rmm::mr::device_memory_resource* mr)
{
  // Compute output sizes
  auto const device_views         = create_strings_device_views(columns, stream);
  auto const& d_views             = std::get<1>(device_views);
  auto const& d_input_offsets     = std::get<2>(device_views);
  auto const& d_partition_offsets = std::get<3>(device_views);
  auto const strings_count        = std::get<4>(device_views);
  auto const total_bytes          = std::get<5>(device_views);
  auto const offsets_count        = strings_count + 1;

  if (strings_count == 0) { return make_empty_strings_column(stream, mr); }

  CUDF_EXPECTS(offsets_count <= std::numeric_limits<size_type>::max(),
               "total number of strings is too large for cudf column");
  CUDF_EXPECTS(total_bytes <= std::numeric_limits<size_type>::max(),
               "total size of strings is too large for cudf column");

  bool const has_nulls =
    std::any_of(columns.begin(), columns.end(), [](auto const& col) { return col.has_nulls(); });

  // create chars column
  auto chars_column =
    make_numeric_column(data_type{type_id::INT8}, total_bytes, mask_state::UNALLOCATED, stream, mr);
  auto d_new_chars = chars_column->mutable_view().data<char>();
  chars_column->set_null_count(0);

  // create offsets column
  auto offsets_column = make_numeric_column(
    data_type{type_id::INT32}, offsets_count, mask_state::UNALLOCATED, stream, mr);
  auto d_new_offsets = offsets_column->mutable_view().data<int32_t>();
  offsets_column->set_null_count(0);

  rmm::device_buffer null_mask{0, stream, mr};
  size_type null_count{};
  if (has_nulls) {
    null_mask =
      cudf::detail::create_null_mask(strings_count, mask_state::UNINITIALIZED, stream, mr);
  }

  {  // Copy offsets columns with single kernel launch
    rmm::device_scalar<size_type> d_valid_count(0);

    constexpr size_type block_size{256};
    cudf::detail::grid_1d config(offsets_count, block_size);
    auto const kernel = has_nulls ? fused_concatenate_string_offset_kernel<block_size, true>
                                  : fused_concatenate_string_offset_kernel<block_size, false>;
    kernel<<<config.num_blocks, config.num_threads_per_block, 0, stream.value()>>>(
      d_views.data().get(),
      d_input_offsets.data().get(),
      d_partition_offsets.data().get(),
      static_cast<size_type>(d_views.size()),
      strings_count,
      d_new_offsets,
      reinterpret_cast<bitmask_type*>(null_mask.data()),
      d_valid_count.data());

    if (has_nulls) { null_count = strings_count - d_valid_count.value(stream); }
  }

  if (total_bytes > 0) {
    // Use a heuristic to guess when the fused kernel will be faster than memcpy
    if (use_fused_kernel_heuristic(has_nulls, total_bytes, columns.size())) {
      // Use single kernel launch to copy chars columns
      constexpr size_type block_size{256};
      cudf::detail::grid_1d config(total_bytes, block_size);
      auto const kernel = fused_concatenate_string_chars_kernel;
      kernel<<<config.num_blocks, config.num_threads_per_block, 0, stream.value()>>>(
        d_views.data().get(),
        d_partition_offsets.data().get(),
        static_cast<size_type>(d_views.size()),
        total_bytes,
        d_new_chars);
    } else {
      // Memcpy each input chars column (more efficient for very large strings)
      for (auto column = columns.begin(); column != columns.end(); ++column) {
        size_type column_size = column->size();
        if (column_size == 0)  // nothing to do
          continue;            // empty column may not have children
        size_type column_offset   = column->offset();
        column_view offsets_child = column->child(strings_column_view::offsets_column_index);
        column_view chars_child   = column->child(strings_column_view::chars_column_index);

        auto d_offsets       = offsets_child.data<int32_t>() + column_offset;
        int32_t bytes_offset = thrust::device_pointer_cast(d_offsets)[0];

        // copy the chars column data
        auto d_chars    = chars_child.data<char>() + bytes_offset;
        size_type bytes = thrust::device_pointer_cast(d_offsets)[column_size] - bytes_offset;
        CUDA_TRY(
          hipMemcpyAsync(d_new_chars, d_chars, bytes, hipMemcpyDeviceToDevice, stream.value()));

        // get ready for the next column
        d_new_chars += bytes;
      }
    }
  }

  return make_strings_column(strings_count,
                             std::move(offsets_column),
                             std::move(chars_column),
                             null_count,
                             std::move(null_mask),
                             stream,
                             mr);
}

}  // namespace detail
}  // namespace strings
}  // namespace cudf
