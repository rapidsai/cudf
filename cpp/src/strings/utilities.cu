/*
 * Copyright (c) 2019-2020, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <strings/char_types/char_cases.h>
#include <strings/char_types/char_flags.h>
#include <strings/utilities.cuh>
#include <strings/utilities.hpp>

#include <cudf/column/column_device_view.cuh>
#include <cudf/column/column_factories.hpp>
#include <cudf/strings/detail/utilities.cuh>
#include <cudf/utilities/error.hpp>

#include <rmm/thrust_rmm_allocator.h>
#include <rmm/cuda_stream_view.hpp>

#include <thrust/transform_reduce.h>
#include <thrust/transform_scan.h>

#include <cstring>

namespace cudf {
namespace strings {
namespace detail {
// Used to build a temporary string_view object from a single host string.
std::unique_ptr<string_view, std::function<void(string_view*)>> string_from_host(
  const char* str, rmm::cuda_stream_view stream)
{
  if (!str) return nullptr;
  auto length = std::strlen(str);

  auto* d_str = new rmm::device_buffer(length, stream);
  CUDA_TRY(hipMemcpyAsync(d_str->data(), str, length, hipMemcpyHostToDevice, stream.value()));
  stream.synchronize();

  auto deleter = [d_str](string_view* sv) { delete d_str; };
  return std::unique_ptr<string_view, decltype(deleter)>{
    new string_view(reinterpret_cast<char*>(d_str->data()), length), deleter};
}

// build a vector of string_view objects from a strings column
rmm::device_vector<string_view> create_string_vector_from_column(cudf::strings_column_view strings,
                                                                 rmm::cuda_stream_view stream)
{
  auto execpol        = rmm::exec_policy(stream);
  auto strings_column = column_device_view::create(strings.parent(), stream);
  auto d_column       = *strings_column;

  auto count = strings.size();
  rmm::device_vector<string_view> strings_vector(count);
  string_view* d_strings = strings_vector.data().get();
  thrust::for_each_n(execpol->on(stream.value()),
                     thrust::make_counting_iterator<size_type>(0),
                     count,
                     [d_column, d_strings] __device__(size_type idx) {
                       if (d_column.is_null(idx))
                         d_strings[idx] = string_view(nullptr, 0);
                       else
                         d_strings[idx] = d_column.element<string_view>(idx);
                     });
  return strings_vector;
}

// build a strings offsets column from a vector of string_views
std::unique_ptr<cudf::column> child_offsets_from_string_vector(
  const rmm::device_vector<string_view>& strings,
  rmm::cuda_stream_view stream,
  rmm::mr::device_memory_resource* mr)
{
  return child_offsets_from_string_iterator(strings.begin(), strings.size(), stream, mr);
}

// build a strings chars column from an vector of string_views
std::unique_ptr<cudf::column> child_chars_from_string_vector(
  const rmm::device_vector<string_view>& strings,
  const int32_t* d_offsets,
  cudf::size_type null_count,
  rmm::cuda_stream_view stream,
  rmm::mr::device_memory_resource* mr)
{
  size_type count = strings.size();
  auto d_strings  = strings.data().get();
  auto execpol    = rmm::exec_policy(stream);
  size_type bytes = thrust::device_pointer_cast(d_offsets)[count];

  // create column
  auto chars_column =
    make_numeric_column(data_type{type_id::INT8}, bytes, mask_state::UNALLOCATED, stream, mr);
  // get it's view
  auto d_chars = chars_column->mutable_view().data<int8_t>();
  thrust::for_each_n(execpol->on(stream.value()),
                     thrust::make_counting_iterator<size_type>(0),
                     count,
                     [d_strings, d_offsets, d_chars] __device__(size_type idx) {
                       string_view const d_str = d_strings[idx];
                       memcpy(d_chars + d_offsets[idx], d_str.data(), d_str.size_bytes());
                     });

  return chars_column;
}

//
std::unique_ptr<column> create_chars_child_column(cudf::size_type strings_count,
                                                  cudf::size_type null_count,
                                                  cudf::size_type total_bytes,
                                                  rmm::cuda_stream_view stream,
                                                  rmm::mr::device_memory_resource* mr)
{
  CUDF_EXPECTS(null_count <= strings_count, "Invalid null count");
  return make_numeric_column(
    data_type{type_id::INT8}, total_bytes, mask_state::UNALLOCATED, stream, mr);
}

//
std::unique_ptr<column> make_empty_strings_column(rmm::cuda_stream_view stream,
                                                  rmm::mr::device_memory_resource* mr)
{
  return std::make_unique<column>(data_type{type_id::STRING},
                                  0,
                                  rmm::device_buffer{0, stream, mr},  // data
                                  rmm::device_buffer{0, stream, mr},
                                  0);  // nulls
}

namespace {
// The device variables are created here to avoid using a singleton that may cause issues
// with RMM initialize/finalize. See PR #3159 for details on this approach.
__device__ character_flags_table_type
  character_codepoint_flags[sizeof(g_character_codepoint_flags)];
__device__ character_cases_table_type character_cases_table[sizeof(g_character_cases_table)];
__device__ special_case_mapping character_special_case_mappings[sizeof(g_special_case_mappings)];

thread_safe_per_context_cache<character_flags_table_type> d_character_codepoint_flags;
thread_safe_per_context_cache<character_cases_table_type> d_character_cases_table;
thread_safe_per_context_cache<special_case_mapping> d_special_case_mappings;

}  // namespace

/**
 * @copydoc cudf::strings::detail::get_character_flags_table
 */
const character_flags_table_type* get_character_flags_table()
{
  return d_character_codepoint_flags.find_or_initialize([&](void) {
    character_flags_table_type* table = nullptr;
    CUDA_TRY(hipMemcpyToSymbol(HIP_SYMBOL(
      character_codepoint_flags), g_character_codepoint_flags, sizeof(g_character_codepoint_flags)));
    CUDA_TRY(hipGetSymbolAddress((void**)&table, character_codepoint_flags));
    return table;
  });
}

/**
 * @copydoc cudf::strings::detail::get_character_cases_table
 */
const character_cases_table_type* get_character_cases_table()
{
  return d_character_cases_table.find_or_initialize([&](void) {
    character_cases_table_type* table = nullptr;
    CUDA_TRY(hipMemcpyToSymbol(HIP_SYMBOL(
      character_cases_table), g_character_cases_table, sizeof(g_character_cases_table)));
    CUDA_TRY(hipGetSymbolAddress((void**)&table, character_cases_table));
    return table;
  });
}

/**
 * @copydoc cudf::strings::detail::get_special_case_mapping_table
 */
const special_case_mapping* get_special_case_mapping_table()
{
  return d_special_case_mappings.find_or_initialize([&](void) {
    special_case_mapping* table = nullptr;
    CUDA_TRY(hipMemcpyToSymbol(HIP_SYMBOL(
      character_special_case_mappings), g_special_case_mappings, sizeof(g_special_case_mappings)));
    CUDA_TRY(hipGetSymbolAddress((void**)&table, character_special_case_mappings));
    return table;
  });
}

}  // namespace detail
}  // namespace strings
}  // namespace cudf
