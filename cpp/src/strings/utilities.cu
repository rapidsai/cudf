/*
 * Copyright (c) 2019-2022, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <strings/char_types/char_cases.h>
#include <strings/char_types/char_flags.h>

#include <cudf/column/column_device_view.cuh>
#include <cudf/column/column_factories.hpp>
#include <cudf/strings/detail/char_tables.hpp>
#include <cudf/strings/detail/utilities.cuh>
#include <cudf/utilities/error.hpp>

#include <rmm/cuda_stream_view.hpp>
#include <rmm/device_uvector.hpp>
#include <rmm/exec_policy.hpp>

#include <thrust/iterator/counting_iterator.h>
#include <thrust/transform.h>

namespace cudf {
namespace strings {
namespace detail {

/**
 * @copydoc create_string_vector_from_column
 */
rmm::device_uvector<string_view> create_string_vector_from_column(
  cudf::strings_column_view const input,
  rmm::cuda_stream_view stream,
  rmm::mr::device_memory_resource* mr)
{
  auto d_strings = column_device_view::create(input.parent(), stream);

  auto strings_vector = rmm::device_uvector<string_view>(input.size(), stream, mr);

  thrust::transform(rmm::exec_policy(stream),
                    thrust::make_counting_iterator<size_type>(0),
                    thrust::make_counting_iterator<size_type>(input.size()),
                    strings_vector.begin(),
                    [d_strings = *d_strings] __device__(size_type idx) {
                      // placeholder for factory function that takes a span of string_views
                      auto const null_string_view = string_view{nullptr, 0};
                      if (d_strings.is_null(idx)) { return null_string_view; }
                      auto const d_str = d_strings.element<string_view>(idx);
                      // special case when the entire column is filled with empty strings:
                      // here the empty d_str may have a d_str.data() == nullptr
                      auto const empty_string_view = string_view{};
                      return d_str.empty() ? empty_string_view : d_str;
                    });

  return strings_vector;
}

std::unique_ptr<column> create_chars_child_column(cudf::size_type total_bytes,
                                                  rmm::cuda_stream_view stream,
                                                  rmm::mr::device_memory_resource* mr)
{
  return make_numeric_column(
    data_type{type_id::INT8}, total_bytes, mask_state::UNALLOCATED, stream, mr);
}

namespace {
// The device variables are created here to avoid using a singleton that may cause issues
// with RMM initialize/finalize. See PR #3159 for details on this approach.
__device__ character_flags_table_type
  character_codepoint_flags[sizeof(g_character_codepoint_flags)];
__device__ character_cases_table_type character_cases_table[sizeof(g_character_cases_table)];
__device__ special_case_mapping character_special_case_mappings[sizeof(g_special_case_mappings)];

thread_safe_per_context_cache<character_flags_table_type> d_character_codepoint_flags;
thread_safe_per_context_cache<character_cases_table_type> d_character_cases_table;
thread_safe_per_context_cache<special_case_mapping> d_special_case_mappings;

}  // namespace

/**
 * @copydoc cudf::strings::detail::get_character_flags_table
 */
const character_flags_table_type* get_character_flags_table()
{
  return d_character_codepoint_flags.find_or_initialize([&](void) {
    character_flags_table_type* table = nullptr;
    CUDF_CUDA_TRY(hipMemcpyToSymbol(HIP_SYMBOL(
      character_codepoint_flags), g_character_codepoint_flags, sizeof(g_character_codepoint_flags)));
    CUDF_CUDA_TRY(hipGetSymbolAddress((void**)&table, character_codepoint_flags));
    return table;
  });
}

/**
 * @copydoc cudf::strings::detail::get_character_cases_table
 */
const character_cases_table_type* get_character_cases_table()
{
  return d_character_cases_table.find_or_initialize([&](void) {
    character_cases_table_type* table = nullptr;
    CUDF_CUDA_TRY(hipMemcpyToSymbol(HIP_SYMBOL(
      character_cases_table), g_character_cases_table, sizeof(g_character_cases_table)));
    CUDF_CUDA_TRY(hipGetSymbolAddress((void**)&table, character_cases_table));
    return table;
  });
}

/**
 * @copydoc cudf::strings::detail::get_special_case_mapping_table
 */
const special_case_mapping* get_special_case_mapping_table()
{
  return d_special_case_mappings.find_or_initialize([&](void) {
    special_case_mapping* table = nullptr;
    CUDF_CUDA_TRY(hipMemcpyToSymbol(HIP_SYMBOL(
      character_special_case_mappings), g_special_case_mappings, sizeof(g_special_case_mappings)));
    CUDF_CUDA_TRY(hipGetSymbolAddress((void**)&table, character_special_case_mappings));
    return table;
  });
}

}  // namespace detail
}  // namespace strings
}  // namespace cudf
