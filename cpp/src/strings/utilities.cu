/*
 * Copyright (c) 2019-2021, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <strings/char_types/char_cases.h>
#include <strings/char_types/char_flags.h>
#include <strings/utilities.cuh>
#include <strings/utilities.hpp>

#include <cudf/column/column_device_view.cuh>
#include <cudf/column/column_factories.hpp>
#include <cudf/detail/get_value.cuh>
#include <cudf/strings/detail/utilities.cuh>
#include <cudf/utilities/error.hpp>

#include <rmm/cuda_stream_view.hpp>
#include <rmm/device_uvector.hpp>
#include <rmm/exec_policy.hpp>

#include <thrust/transform_reduce.h>
#include <thrust/transform_scan.h>

#include <cstring>

namespace cudf {
namespace strings {
namespace detail {
// Used to build a temporary string_view object from a single host string.
std::unique_ptr<string_view, std::function<void(string_view*)>> string_from_host(
  const char* str, rmm::cuda_stream_view stream)
{
  if (!str) return nullptr;
  auto length = std::strlen(str);

  auto* d_str = new rmm::device_buffer(length, stream);
  CUDA_TRY(hipMemcpyAsync(d_str->data(), str, length, hipMemcpyHostToDevice, stream.value()));
  stream.synchronize();

  auto deleter = [d_str](string_view* sv) { delete d_str; };
  return std::unique_ptr<string_view, decltype(deleter)>{
    new string_view(reinterpret_cast<char*>(d_str->data()), length), deleter};
}

/**
 * @copydoc create_string_vector_from_column
 */
rmm::device_uvector<string_view> create_string_vector_from_column(cudf::strings_column_view strings,
                                                                  rmm::cuda_stream_view stream)
{
  auto strings_column = column_device_view::create(strings.parent(), stream);
  auto d_column       = *strings_column;

  rmm::device_uvector<string_view> strings_vector(strings.size(), stream);
  string_view* d_strings = strings_vector.data();
  thrust::for_each_n(rmm::exec_policy(stream),
                     thrust::make_counting_iterator<size_type>(0),
                     strings.size(),
                     [d_column, d_strings] __device__(size_type idx) {
                       if (d_column.is_null(idx))
                         d_strings[idx] = string_view(nullptr, 0);
                       else
                         d_strings[idx] = d_column.element<string_view>(idx);
                     });
  return strings_vector;
}

/**
 * @copydoc child_offsets_from_string_vector
 */
std::unique_ptr<cudf::column> child_offsets_from_string_vector(
  cudf::detail::device_span<string_view> strings,
  rmm::cuda_stream_view stream,
  rmm::mr::device_memory_resource* mr)
{
  return child_offsets_from_string_iterator(strings.begin(), strings.size(), stream, mr);
}

/**
 * @copydoc child_chars_from_string_vector
 */
std::unique_ptr<cudf::column> child_chars_from_string_vector(
  cudf::detail::device_span<string_view> strings,
  column_view const& offsets,
  rmm::cuda_stream_view stream,
  rmm::mr::device_memory_resource* mr)
{
  auto const d_strings = strings.data();
  auto const bytes     = cudf::detail::get_value<int32_t>(offsets, strings.size(), stream);
  auto const d_offsets = offsets.data<int32_t>();

  // create column
  auto chars_column =
    make_numeric_column(data_type{type_id::INT8}, bytes, mask_state::UNALLOCATED, stream, mr);
  // get it's view
  auto d_chars = chars_column->mutable_view().data<char>();
  thrust::for_each_n(rmm::exec_policy(stream),
                     thrust::make_counting_iterator<size_type>(0),
                     strings.size(),
                     [d_strings, d_offsets, d_chars] __device__(size_type idx) {
                       string_view const d_str = d_strings[idx];
                       memcpy(d_chars + d_offsets[idx], d_str.data(), d_str.size_bytes());
                     });

  return chars_column;
}

//
std::unique_ptr<column> create_chars_child_column(cudf::size_type strings_count,
                                                  cudf::size_type null_count,
                                                  cudf::size_type total_bytes,
                                                  rmm::cuda_stream_view stream,
                                                  rmm::mr::device_memory_resource* mr)
{
  CUDF_EXPECTS(null_count <= strings_count, "Invalid null count");
  return make_numeric_column(
    data_type{type_id::INT8}, total_bytes, mask_state::UNALLOCATED, stream, mr);
}

//
std::unique_ptr<column> make_empty_strings_column(rmm::cuda_stream_view stream,
                                                  rmm::mr::device_memory_resource* mr)
{
  return std::make_unique<column>(data_type{type_id::STRING},
                                  0,
                                  rmm::device_buffer{0, stream, mr},  // data
                                  rmm::device_buffer{0, stream, mr},
                                  0);  // nulls
}

namespace {
// The device variables are created here to avoid using a singleton that may cause issues
// with RMM initialize/finalize. See PR #3159 for details on this approach.
__device__ character_flags_table_type
  character_codepoint_flags[sizeof(g_character_codepoint_flags)];
__device__ character_cases_table_type character_cases_table[sizeof(g_character_cases_table)];
__device__ special_case_mapping character_special_case_mappings[sizeof(g_special_case_mappings)];

thread_safe_per_context_cache<character_flags_table_type> d_character_codepoint_flags;
thread_safe_per_context_cache<character_cases_table_type> d_character_cases_table;
thread_safe_per_context_cache<special_case_mapping> d_special_case_mappings;

}  // namespace

/**
 * @copydoc cudf::strings::detail::get_character_flags_table
 */
const character_flags_table_type* get_character_flags_table()
{
  return d_character_codepoint_flags.find_or_initialize([&](void) {
    character_flags_table_type* table = nullptr;
    CUDA_TRY(hipMemcpyToSymbol(HIP_SYMBOL(
      character_codepoint_flags), g_character_codepoint_flags, sizeof(g_character_codepoint_flags)));
    CUDA_TRY(hipGetSymbolAddress((void**)&table, character_codepoint_flags));
    return table;
  });
}

/**
 * @copydoc cudf::strings::detail::get_character_cases_table
 */
const character_cases_table_type* get_character_cases_table()
{
  return d_character_cases_table.find_or_initialize([&](void) {
    character_cases_table_type* table = nullptr;
    CUDA_TRY(hipMemcpyToSymbol(HIP_SYMBOL(
      character_cases_table), g_character_cases_table, sizeof(g_character_cases_table)));
    CUDA_TRY(hipGetSymbolAddress((void**)&table, character_cases_table));
    return table;
  });
}

/**
 * @copydoc cudf::strings::detail::get_special_case_mapping_table
 */
const special_case_mapping* get_special_case_mapping_table()
{
  return d_special_case_mappings.find_or_initialize([&](void) {
    special_case_mapping* table = nullptr;
    CUDA_TRY(hipMemcpyToSymbol(HIP_SYMBOL(
      character_special_case_mappings), g_special_case_mappings, sizeof(g_special_case_mappings)));
    CUDA_TRY(hipGetSymbolAddress((void**)&table, character_special_case_mappings));
    return table;
  });
}

}  // namespace detail
}  // namespace strings
}  // namespace cudf
