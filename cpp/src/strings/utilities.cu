/*
 * Copyright (c) 2019, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <cstring>
#include <cudf/column/column_device_view.cuh>
#include <cudf/column/column_factories.hpp>
#include <cudf/utilities/error.hpp>
#include <unordered_map>
#include "./utilities.cuh"
#include "./utilities.hpp"
#include "char_types/char_cases.h"
#include "char_types/char_flags.h"

#include <rmm/thrust_rmm_allocator.h>
#include <thrust/transform_reduce.h>
#include <thrust/transform_scan.h>
#include <mutex>

namespace cudf {
namespace strings {
namespace detail {
// Used to build a temporary string_view object from a single host string.
std::unique_ptr<string_view, std::function<void(string_view*)>> string_from_host(
  const char* str, hipStream_t stream)
{
  if (!str) return nullptr;
  auto length = std::strlen(str);

  auto* d_str = new rmm::device_buffer(length, stream);
  CUDA_TRY(hipMemcpyAsync(d_str->data(), str, length, hipMemcpyHostToDevice, stream));
  CUDA_TRY(hipStreamSynchronize(stream));

  auto deleter = [d_str](string_view* sv) { delete d_str; };
  return std::unique_ptr<string_view, decltype(deleter)>{
    new string_view(reinterpret_cast<char*>(d_str->data()), length), deleter};
}

// build a vector of string_view objects from a strings column
rmm::device_vector<string_view> create_string_vector_from_column(cudf::strings_column_view strings,
                                                                 hipStream_t stream)
{
  auto execpol        = rmm::exec_policy(stream);
  auto strings_column = column_device_view::create(strings.parent(), stream);
  auto d_column       = *strings_column;

  auto count = strings.size();
  rmm::device_vector<string_view> strings_vector(count);
  string_view* d_strings = strings_vector.data().get();
  thrust::for_each_n(execpol->on(stream),
                     thrust::make_counting_iterator<size_type>(0),
                     count,
                     [d_column, d_strings] __device__(size_type idx) {
                       if (d_column.is_null(idx))
                         d_strings[idx] = string_view(nullptr, 0);
                       else
                         d_strings[idx] = d_column.element<string_view>(idx);
                     });
  return strings_vector;
}

// build a strings offsets column from a vector of string_views
std::unique_ptr<cudf::column> child_offsets_from_string_vector(
  const rmm::device_vector<string_view>& strings,
  rmm::mr::device_memory_resource* mr,
  hipStream_t stream)
{
  auto transformer = [] __device__(string_view v) { return v.size_bytes(); };
  auto begin       = thrust::make_transform_iterator(strings.begin(), transformer);
  return make_offsets_child_column(begin, begin + strings.size(), mr, stream);
}

// build a strings chars column from an vector of string_views
std::unique_ptr<cudf::column> child_chars_from_string_vector(
  const rmm::device_vector<string_view>& strings,
  const int32_t* d_offsets,
  cudf::size_type null_count,
  rmm::mr::device_memory_resource* mr,
  hipStream_t stream)
{
  size_type count = strings.size();
  auto d_strings  = strings.data().get();
  auto execpol    = rmm::exec_policy(stream);
  size_type bytes = thrust::device_pointer_cast(d_offsets)[count];

  // create column
  auto chars_column =
    make_numeric_column(data_type{type_id::INT8}, bytes, mask_state::UNALLOCATED, stream, mr);
  // get it's view
  auto d_chars = chars_column->mutable_view().data<int8_t>();
  thrust::for_each_n(execpol->on(stream),
                     thrust::make_counting_iterator<size_type>(0),
                     count,
                     [d_strings, d_offsets, d_chars] __device__(size_type idx) {
                       string_view const d_str = d_strings[idx];
                       memcpy(d_chars + d_offsets[idx], d_str.data(), d_str.size_bytes());
                     });

  return chars_column;
}

//
std::unique_ptr<column> create_chars_child_column(cudf::size_type strings_count,
                                                  cudf::size_type null_count,
                                                  cudf::size_type total_bytes,
                                                  rmm::mr::device_memory_resource* mr,
                                                  hipStream_t stream)
{
  CUDF_EXPECTS(null_count <= strings_count, "Invalid null count");
  return make_numeric_column(
    data_type{type_id::INT8}, total_bytes, mask_state::UNALLOCATED, stream, mr);
}

//
std::unique_ptr<column> make_empty_strings_column(rmm::mr::device_memory_resource* mr,
                                                  hipStream_t stream)
{
  return std::make_unique<column>(data_type{type_id::STRING},
                                  0,
                                  rmm::device_buffer{0, stream, mr},  // data
                                  rmm::device_buffer{0, stream, mr},
                                  0);  // nulls
}

namespace {
// The device variables are created here to avoid using a singleton that may cause issues
// with RMM initialize/finalize. See PR #3159 for details on this approach.
__device__ character_flags_table_type
  character_codepoint_flags[sizeof(g_character_codepoint_flags)];
__device__ character_cases_table_type character_cases_table[sizeof(g_character_cases_table)];
__device__ special_case_mapping character_special_case_mappings[sizeof(g_special_case_mappings)];

// This template is a thin wrapper around per-context singleton objects.
// It maintains a single object for each CUDA context.
template <typename TableType>
class per_context_cache {
 public:
  // Find an object cached for a current CUDA context.
  // If there is no object available in the cache, it calls the initializer
  // `init` to create a new one and cache it for later uses.
  template <typename Initializer>
  TableType* find_or_initialize(const Initializer& init)
  {
    hipCtx_t c;
    hipCtxGetCurrent(&c);
    auto finder = cache_.find(c);
    if (finder == cache_.end()) {
      TableType* result = init();
      cache_[c]         = result;
      return result;
    } else
      return finder->second;
  }

 private:
  std::unordered_map<hipCtx_t, TableType*> cache_;
};

// This template is a thread-safe version of per_context_cache.
template <typename TableType>
class thread_safe_per_context_cache : public per_context_cache<TableType> {
 public:
  template <typename Initializer>
  TableType* find_or_initialize(const Initializer& init)
  {
    std::lock_guard<std::mutex> guard(mutex);
    return per_context_cache<TableType>::find_or_initialize(init);
  }

 private:
  std::mutex mutex;
};

thread_safe_per_context_cache<character_flags_table_type> d_character_codepoint_flags;
thread_safe_per_context_cache<character_cases_table_type> d_character_cases_table;
thread_safe_per_context_cache<special_case_mapping> d_special_case_mappings;

}  // namespace

/**
 * @copydoc cudf::strings::detail::get_character_flags_table
 */
const character_flags_table_type* get_character_flags_table()
{
  return d_character_codepoint_flags.find_or_initialize([&](void) {
    character_flags_table_type* table = nullptr;
    CUDA_TRY(hipMemcpyToSymbol(HIP_SYMBOL(
      character_codepoint_flags), g_character_codepoint_flags, sizeof(g_character_codepoint_flags)));
    CUDA_TRY(hipGetSymbolAddress((void**)&table, character_codepoint_flags));
    return table;
  });
}

/**
 * @copydoc cudf::strings::detail::get_character_cases_table
 */
const character_cases_table_type* get_character_cases_table()
{
  return d_character_cases_table.find_or_initialize([&](void) {
    character_cases_table_type* table = nullptr;
    CUDA_TRY(hipMemcpyToSymbol(HIP_SYMBOL(
      character_cases_table), g_character_cases_table, sizeof(g_character_cases_table)));
    CUDA_TRY(hipGetSymbolAddress((void**)&table, character_cases_table));
    return table;
  });
}

/**
 * @copydoc cudf::strings::detail::get_special_case_mapping_table
 */
const special_case_mapping* get_special_case_mapping_table()
{
  return d_special_case_mappings.find_or_initialize([&](void) {
    special_case_mapping* table = nullptr;
    CUDA_TRY(hipMemcpyToSymbol(HIP_SYMBOL(
      character_special_case_mappings), g_special_case_mappings, sizeof(g_special_case_mappings)));
    CUDA_TRY(hipGetSymbolAddress((void**)&table, character_special_case_mappings));
    return table;
  });
}

}  // namespace detail
}  // namespace strings
}  // namespace cudf
