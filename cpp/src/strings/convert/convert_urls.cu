#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019-2024, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <cudf/column/column_device_view.cuh>
#include <cudf/column/column_factories.hpp>
#include <cudf/detail/null_mask.hpp>
#include <cudf/detail/nvtx/ranges.hpp>
#include <cudf/detail/offsets_iterator_factory.cuh>
#include <cudf/detail/utilities/cuda.cuh>
#include <cudf/detail/utilities/integer_utils.hpp>
#include <cudf/strings/convert/convert_urls.hpp>
#include <cudf/strings/detail/strings_children.cuh>
#include <cudf/strings/detail/utilities.cuh>
#include <cudf/strings/string_view.cuh>
#include <cudf/strings/strings_column_view.hpp>
#include <cudf/types.hpp>
#include <cudf/utilities/default_stream.hpp>

#include <rmm/cuda_stream_view.hpp>
#include <rmm/device_uvector.hpp>
#include <rmm/resource_ref.hpp>

#include <hipcub/hipcub.hpp>

namespace cudf {
namespace strings {
namespace detail {
namespace {
//
// This is the functor for the url_encode() method below.
// Specific requirements are documented in custrings issue #321.
// In summary it converts mostly non-ascii characters and control characters into UTF-8 hex
// characters prefixed with '%'. For example, the space character must be converted to characters
// '%20' where the '20' indicates the hex value for space in UTF-8. Likewise, multi-byte characters
// are converted to multiple hex characters. For example, the é character is converted to characters
// '%C3%A9' where 'C3A9' is the UTF-8 bytes xc3a9 for this character.
//
struct url_encoder_fn {
  column_device_view const d_strings;
  size_type* d_sizes{};
  char* d_chars{};
  cudf::detail::input_offsetalator d_offsets;

  // utility to create 2-byte hex characters from single binary byte
  __device__ void byte_to_hex(uint8_t byte, char* hex)
  {
    hex[0] = '0';
    if (byte >= 16) {
      uint8_t hibyte = byte / 16;
      hex[0]         = hibyte < 10 ? '0' + hibyte : 'A' + (hibyte - 10);
      byte           = byte - (hibyte * 16);
    }
    hex[1] = byte < 10 ? '0' + byte : 'A' + (byte - 10);
  }

  __device__ bool should_not_url_encode(char ch)
  {
    return (
      (ch >= '0' && ch <= '9') ||  // these are the characters
      (ch >= 'A' && ch <= 'Z') ||  // that are not to be url encoded
      (ch >= 'a' &&
       ch <= 'z') ||  // reference: docs.python.org/3/library/urllib.parse.html#urllib.parse.quote
      (ch == '.') ||
      (ch == '_') || (ch == '~') || (ch == '-'));
  }

  // main part of the functor the performs the url-encoding
  __device__ void operator()(size_type idx)
  {
    if (d_strings.is_null(idx)) {
      if (!d_chars) { d_sizes[idx] = 0; }
      return;
    }

    string_view d_str = d_strings.element<string_view>(idx);
    //
    char* out_ptr    = d_chars ? d_chars + d_offsets[idx] : nullptr;
    size_type nbytes = 0;
    char hex[2];  // two-byte hex max
    for (auto itr = d_str.begin(); itr != d_str.end(); ++itr) {
      auto ch = *itr;
      if (ch < 128) {
        if (should_not_url_encode(static_cast<char>(ch))) {
          nbytes++;
          if (out_ptr) out_ptr = copy_and_increment(out_ptr, d_str.data() + itr.byte_offset(), 1);
        } else  // url-encode everything else
        {
          nbytes += 3;
          if (out_ptr) {
            out_ptr = copy_and_increment(out_ptr, "%", 1);  // add the '%' prefix
            byte_to_hex(static_cast<uint8_t>(ch), hex);     // convert to 2 hex chars
            out_ptr = copy_and_increment(out_ptr, hex, 2);  // add them to the output
          }
        }
      } else  // these are to be utf-8 url-encoded
      {
        uint8_t char_bytes[4];  // holds utf-8 bytes for one character
        size_type char_width = from_char_utf8(ch, reinterpret_cast<char*>(char_bytes));
        nbytes += char_width * 3;  // '%' plus 2 hex chars per byte (example: é is %C3%A9)
        // process each byte in this current character
        for (size_type chidx = 0; out_ptr && (chidx < char_width); ++chidx) {
          out_ptr = copy_and_increment(out_ptr, "%", 1);  // add '%' prefix
          byte_to_hex(char_bytes[chidx], hex);            // convert to 2 hex chars
          out_ptr = copy_and_increment(out_ptr, hex, 2);  // add them to the output
        }
      }
    }
    if (!d_chars) { d_sizes[idx] = nbytes; }
  }
};

}  // namespace

//
std::unique_ptr<column> url_encode(strings_column_view const& input,
                                   rmm::cuda_stream_view stream,
                                   rmm::device_async_resource_ref mr)
{
  if (input.is_empty()) return make_empty_column(type_id::STRING);

  auto d_column = column_device_view::create(input.parent(), stream);

  auto [offsets_column, chars] =
    make_strings_children(url_encoder_fn{*d_column}, input.size(), stream, mr);

  return make_strings_column(input.size(),
                             std::move(offsets_column),
                             chars.release(),
                             input.null_count(),
                             cudf::detail::copy_bitmask(input.parent(), stream, mr));
}

}  // namespace detail

// external API
std::unique_ptr<column> url_encode(strings_column_view const& input,
                                   rmm::cuda_stream_view stream,
                                   rmm::device_async_resource_ref mr)
{
  CUDF_FUNC_RANGE();
  return detail::url_encode(input, stream, mr);
}

namespace detail {
namespace {

// utility to convert a hex char into a single byte
constexpr uint8_t hex_char_to_byte(char ch)
{
  if (ch >= '0' && ch <= '9') return (ch - '0');
  if (ch >= 'A' && ch <= 'F') return (ch - 'A' + 10);  // in hex A=10,B=11,...,F=15
  if (ch >= 'a' && ch <= 'f') return (ch - 'a' + 10);  // same for lower case
  return 0;
}

constexpr bool is_hex_digit(char ch)
{
  return (ch >= '0' && ch <= '9') || (ch >= 'A' && ch <= 'F') || (ch >= 'a' && ch <= 'f');
}

__forceinline__ __device__ bool is_escape_char(char const* const ptr)
{
  return (ptr[0] == '%' && is_hex_digit(ptr[1]) && is_hex_digit(ptr[2]));
}

// helper function for converting an escaped sequence starting at `ptr` to a single byte
__forceinline__ __device__ char escaped_sequence_to_byte(char const* const ptr)
{
  return (hex_char_to_byte(ptr[1]) << 4) | hex_char_to_byte(ptr[2]);
}

/**
 * @brief Count the number of characters of each string after URL decoding.
 *
 * @tparam num_warps_per_threadblock Number of warps in a threadblock. This template argument must
 * match the launch configuration, i.e. the kernel must be launched with
 * `num_warps_per_threadblock * cudf::detail::warp_size` threads per threadblock.
 * @tparam char_block_size Number of characters which will be loaded into the shared memory at a
 * time.
 *
 * @param[in] in_strings Input string column.
 * @param[out] out_counts Number of characters in each decode URL.
 */
template <size_type num_warps_per_threadblock, size_type char_block_size>
CUDF_KERNEL void url_decode_char_counter(column_device_view const in_strings,
                                         size_type* const out_counts)
{
  constexpr int halo_size = 2;
  __shared__ char temporary_buffer[num_warps_per_threadblock][char_block_size + halo_size];
  __shared__ typename hipcub::WarpReduce<int8_t>::TempStorage cub_storage[num_warps_per_threadblock];

  auto const global_thread_id =
    cudf::detail::grid_1d::global_thread_id<num_warps_per_threadblock * cudf::detail::warp_size>();
  auto const global_warp_id = static_cast<size_type>(global_thread_id / cudf::detail::warp_size);
  auto const local_warp_id  = static_cast<size_type>(threadIdx.x / cudf::detail::warp_size);
  auto const warp_lane      = static_cast<size_type>(threadIdx.x % cudf::detail::warp_size);
  auto const nwarps     = static_cast<size_type>(gridDim.x * blockDim.x / cudf::detail::warp_size);
  char* in_chars_shared = temporary_buffer[local_warp_id];

  // Loop through strings, and assign each string to a warp.
  for (thread_index_type tidx = global_warp_id; tidx < in_strings.size(); tidx += nwarps) {
    auto const row_idx = static_cast<size_type>(tidx);
    if (in_strings.is_null(row_idx)) {
      out_counts[row_idx] = 0;
      continue;
    }

    auto const in_string     = in_strings.element<string_view>(row_idx);
    auto const in_chars      = in_string.data();
    auto const string_length = in_string.size_bytes();
    auto const nblocks       = cudf::util::div_rounding_up_unsafe(string_length, char_block_size);
    size_type escape_char_count = 0;

    for (size_type block_idx = 0; block_idx < nblocks; block_idx++) {
      auto const string_length_block =
        std::min(char_block_size, string_length - char_block_size * block_idx);

      // Each warp collectively loads input characters of the current block to the shared memory.
      // When testing whether a location is the start of an escaped character, we need to access
      // the current location as well as the next two locations. To avoid branches, two halo cells
      // are added after the end of the block. If the cell is beyond the end of the string, 0s are
      // filled in to make sure the last two characters of the string are not the start of an
      // escaped sequence.
      for (auto char_idx = warp_lane; char_idx < string_length_block + halo_size;
           char_idx += cudf::detail::warp_size) {
        auto const in_idx         = block_idx * char_block_size + char_idx;
        in_chars_shared[char_idx] = in_idx < string_length ? in_chars[in_idx] : 0;
      }

      __syncwarp();

      // `char_idx_start` represents the start character index of the current warp.
      for (size_type char_idx_start = 0; char_idx_start < string_length_block;
           char_idx_start += cudf::detail::warp_size) {
        auto const char_idx = char_idx_start + warp_lane;
        int8_t const is_ichar_escape_char =
          (char_idx < string_length_block && is_escape_char(in_chars_shared + char_idx)) ? 1 : 0;

        // Warp-wise reduction to calculate the number of escape characters.
        // All threads in the warp participate in the reduction, even if `char_idx` is beyond
        // `string_length_block`.
        int8_t const total_escape_char =
          hipcub::WarpReduce<int8_t>(cub_storage[local_warp_id]).Sum(is_ichar_escape_char);

        if (warp_lane == 0) { escape_char_count += total_escape_char; }

        __syncwarp();
      }
    }
    // URL decoding replaces 3 bytes with 1 for each escape character.
    if (warp_lane == 0) { out_counts[row_idx] = string_length - escape_char_count * 2; }
  }
}

/**
 * @brief Decode and copy from the input string column to the output char buffer.
 *
 * @tparam num_warps_per_threadblock Number of warps in a threadblock. This template argument must
 * match the launch configuration, i.e. the kernel must be launched with
 * `num_warps_per_threadblock * cudf::detail::warp_size` threads per threadblock.
 * @tparam char_block_size Number of characters which will be loaded into the shared memory at a
 * time.
 *
 * @param[in] in_strings Input string column.
 * @param[out] out_chars Character buffer for the output string column.
 * @param[in] out_offsets Offset value of each string associated with `out_chars`.
 */
template <size_type num_warps_per_threadblock, size_type char_block_size>
CUDF_KERNEL void url_decode_char_replacer(column_device_view const in_strings,
                                          char* const out_chars,
                                          cudf::detail::input_offsetalator const out_offsets)
{
  constexpr int halo_size = 2;
  __shared__ char temporary_buffer[num_warps_per_threadblock][char_block_size + halo_size * 2];
  __shared__ typename hipcub::WarpScan<int8_t>::TempStorage cub_storage[num_warps_per_threadblock];
  __shared__ size_type out_idx[num_warps_per_threadblock];

  auto const global_thread_id =
    cudf::detail::grid_1d::global_thread_id<num_warps_per_threadblock * cudf::detail::warp_size>();
  auto const global_warp_id = static_cast<size_type>(global_thread_id / cudf::detail::warp_size);
  auto const local_warp_id  = static_cast<size_type>(threadIdx.x / cudf::detail::warp_size);
  auto const warp_lane      = static_cast<size_type>(threadIdx.x % cudf::detail::warp_size);
  auto const nwarps     = static_cast<size_type>(gridDim.x * blockDim.x / cudf::detail::warp_size);
  char* in_chars_shared = temporary_buffer[local_warp_id];

  // Loop through strings, and assign each string to a warp
  for (thread_index_type tidx = global_warp_id; tidx < in_strings.size(); tidx += nwarps) {
    auto const row_idx = static_cast<size_type>(tidx);
    if (in_strings.is_null(row_idx)) continue;

    auto const in_string     = in_strings.element<string_view>(row_idx);
    auto const in_chars      = in_string.data();
    auto const string_length = in_string.size_bytes();
    auto out_chars_string    = out_chars + out_offsets[row_idx];
    auto const nblocks       = cudf::util::div_rounding_up_unsafe(string_length, char_block_size);

    // Use the last thread of the warp to initialize `out_idx` to 0.
    if (warp_lane == cudf::detail::warp_size - 1) { out_idx[local_warp_id] = 0; }

    for (size_type block_idx = 0; block_idx < nblocks; block_idx++) {
      auto const string_length_block =
        std::min(char_block_size, string_length - char_block_size * block_idx);

      // Each warp collectively loads input characters of the current block to shared memory.
      // Two halo cells before and after the block are added. The halo cells are used to test
      // whether the current location as well as the previous two locations are escape characters,
      // without branches.
      for (auto char_idx = warp_lane; char_idx < string_length_block + halo_size * 2;
           char_idx += cudf::detail::warp_size) {
        auto const in_idx         = block_idx * char_block_size + char_idx - halo_size;
        in_chars_shared[char_idx] = in_idx >= 0 && in_idx < string_length ? in_chars[in_idx] : 0;
      }

      __syncwarp();

      // `char_idx_start` represents the start character index of the current warp.
      for (size_type char_idx_start = 0; char_idx_start < string_length_block;
           char_idx_start += cudf::detail::warp_size) {
        auto const char_idx = char_idx_start + warp_lane;
        // If the current character is part of an escape sequence starting at the previous two
        // locations, the thread with the starting location should output the escaped character, and
        // the current thread should not output a character.
        int8_t const out_size =
          (char_idx >= string_length_block || is_escape_char(in_chars_shared + char_idx) ||
           is_escape_char(in_chars_shared + char_idx + 1))
            ? 0
            : 1;

        // Warp-wise prefix sum to establish output location of the current thread.
        // All threads in the warp participate in the prefix sum, even if `char_idx` is beyond
        // `string_length_block`.
        int8_t out_offset;
        hipcub::WarpScan<int8_t>(cub_storage[local_warp_id]).ExclusiveSum(out_size, out_offset);

        if (out_size == 1) {
          char const* const ch_ptr = in_chars_shared + char_idx + halo_size;
          char const ch =
            is_escape_char(ch_ptr)
              ?
              // If the current location is the start of an escape sequence, load and decode.
              escaped_sequence_to_byte(ch_ptr)
              :
              // If the current location is not the start of an escape sequence, load directly.
              *ch_ptr;
          out_chars_string[out_idx[local_warp_id] + out_offset] = ch;
        }

        if (warp_lane == cudf::detail::warp_size - 1) {
          out_idx[local_warp_id] += (out_offset + out_size);
        }

        __syncwarp();
      }
    }
  }
}

}  // namespace

//
std::unique_ptr<column> url_decode(strings_column_view const& strings,
                                   rmm::cuda_stream_view stream,
                                   rmm::device_async_resource_ref mr)
{
  size_type strings_count = strings.size();
  if (strings_count == 0) return make_empty_column(type_id::STRING);

  constexpr size_type num_warps_per_threadblock = 4;
  constexpr size_type threadblock_size = num_warps_per_threadblock * cudf::detail::warp_size;
  constexpr size_type char_block_size  = 256;
  auto const num_threadblocks =
    std::min(65536, cudf::util::div_rounding_up_unsafe(strings_count, num_warps_per_threadblock));

  auto const d_strings = column_device_view::create(strings.parent(), stream);

  // build offsets column by computing the output row sizes and scanning the results
  auto row_sizes = rmm::device_uvector<size_type>(strings_count, stream);
  url_decode_char_counter<num_warps_per_threadblock, char_block_size>
    <<<num_threadblocks, threadblock_size, 0, stream.value()>>>(*d_strings, row_sizes.data());
  // performs scan on the sizes and builds the appropriate offsets column
  auto [offsets_column, out_chars_bytes] = cudf::strings::detail::make_offsets_child_column(
    row_sizes.begin(), row_sizes.end(), stream, mr);

  // create the chars column
  rmm::device_uvector<char> chars(out_chars_bytes, stream, mr);
  auto d_out_chars = chars.data();
  auto const offsets =
    cudf::detail::offsetalator_factory::make_input_iterator(offsets_column->view());

  // decode and copy the characters from the input column to the output column
  url_decode_char_replacer<num_warps_per_threadblock, char_block_size>
    <<<num_threadblocks, threadblock_size, 0, stream.value()>>>(*d_strings, d_out_chars, offsets);

  // copy null mask
  rmm::device_buffer null_mask = cudf::detail::copy_bitmask(strings.parent(), stream, mr);

  return make_strings_column(strings_count,
                             std::move(offsets_column),
                             chars.release(),
                             strings.null_count(),
                             std::move(null_mask));
}

}  // namespace detail

// external API

std::unique_ptr<column> url_decode(strings_column_view const& input,
                                   rmm::cuda_stream_view stream,
                                   rmm::device_async_resource_ref mr)
{
  CUDF_FUNC_RANGE();
  return detail::url_decode(input, stream, mr);
}

}  // namespace strings
}  // namespace cudf
