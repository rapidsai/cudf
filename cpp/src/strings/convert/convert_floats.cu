#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019-2020, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <cudf/column/column_device_view.cuh>
#include <cudf/column/column_factories.hpp>
#include <cudf/detail/null_mask.hpp>
#include <cudf/detail/nvtx/ranges.hpp>
#include <cudf/strings/convert/convert_floats.hpp>
#include <cudf/strings/detail/converters.hpp>
#include <cudf/strings/detail/utilities.hpp>
#include <cudf/strings/string_view.cuh>
#include <cudf/strings/strings_column_view.hpp>
#include <cudf/utilities/traits.hpp>
#include <cudf/utilities/type_dispatcher.hpp>
#include <strings/utilities.cuh>

#include <rmm/cuda_stream_view.hpp>
#include <rmm/exec_policy.hpp>

#include <thrust/iterator/counting_iterator.h>
#include <thrust/transform.h>

#include <cmath>
#include <limits>

namespace cudf {
namespace strings {
namespace detail {
namespace {
/**
 * @brief This function converts the given string into a
 * floating point double value.
 *
 * This will also map strings containing "NaN", "Inf" and "-Inf"
 * to the appropriate float values.
 *
 * This function will also handle scientific notation format.
 */
__device__ inline double stod(string_view const& d_str)
{
  const char* in_ptr = d_str.data();
  const char* end    = in_ptr + d_str.size_bytes();
  if (end == in_ptr) return 0.0;
  // special strings
  if (d_str.compare("NaN", 3) == 0) return std::numeric_limits<double>::quiet_NaN();
  if (d_str.compare("Inf", 3) == 0) return std::numeric_limits<double>::infinity();
  if (d_str.compare("-Inf", 4) == 0) return -std::numeric_limits<double>::infinity();
  double sign = 1.0;
  if (*in_ptr == '-' || *in_ptr == '+') {
    sign = (*in_ptr == '-' ? -1 : 1);
    ++in_ptr;
  }
  unsigned long max_mantissa = 0x0FFFFFFFFFFFFF;
  unsigned long digits       = 0;
  int exp_off                = 0;
  bool decimal               = false;
  while (in_ptr < end) {
    char ch = *in_ptr;
    if (ch == '.') {
      decimal = true;
      ++in_ptr;
      continue;
    }
    if (ch < '0' || ch > '9') break;
    if (digits > max_mantissa)
      exp_off += (int)!decimal;
    else {
      digits = (digits * 10L) + (unsigned long)(ch - '0');
      if (digits > max_mantissa) {
        digits = digits / 10L;
        exp_off += (int)!decimal;
      } else
        exp_off -= (int)decimal;
    }
    ++in_ptr;
  }
  // check for exponent char
  int exp_ten  = 0;
  int exp_sign = 1;
  if (in_ptr < end) {
    char ch = *in_ptr++;
    if (ch == 'e' || ch == 'E') {
      if (in_ptr < end) {
        ch = *in_ptr;
        if (ch == '-' || ch == '+') {
          exp_sign = (ch == '-' ? -1 : 1);
          ++in_ptr;
        }
        while (in_ptr < end) {
          ch = *in_ptr++;
          if (ch < '0' || ch > '9') break;
          exp_ten = (exp_ten * 10) + (int)(ch - '0');
        }
      }
    }
  }
  exp_ten *= exp_sign;
  exp_ten += exp_off;
  if (exp_ten > 308)
    return sign > 0 ? std::numeric_limits<double>::infinity()
                    : -std::numeric_limits<double>::infinity();
  else if (exp_ten < -308)
    return 0.0;
  // using exp10() since the pow(10.0,exp_ten) function is
  // very inaccurate in 10.2: http://nvbugs/2971187
  double value = static_cast<double>(digits) * exp10(static_cast<double>(exp_ten));
  return (value * sign);
}

/**
 * @brief Converts strings column entries into floats.
 *
 * Used by the dispatch method to convert to different float types.
 */
template <typename FloatType>
struct string_to_float_fn {
  const column_device_view strings_column;  // strings to convert

  __device__ FloatType operator()(size_type idx)
  {
    if (strings_column.is_null(idx)) return static_cast<FloatType>(0);
    // the cast to FloatType will create predictable results
    // for floats that are larger than the FloatType can hold
    return static_cast<FloatType>(stod(strings_column.element<string_view>(idx)));
  }
};

/**
 * @brief The dispatch functions for converting strings to floats.
 *
 * The output_column is expected to be one of the float types only.
 */
struct dispatch_to_floats_fn {
  template <typename FloatType,
            std::enable_if_t<std::is_floating_point<FloatType>::value>* = nullptr>
  void operator()(column_device_view const& strings_column,
                  mutable_column_view& output_column,
                  rmm::cuda_stream_view stream) const
  {
    auto d_results = output_column.data<FloatType>();
    thrust::transform(rmm::exec_policy(stream),
                      thrust::make_counting_iterator<size_type>(0),
                      thrust::make_counting_iterator<size_type>(strings_column.size()),
                      d_results,
                      string_to_float_fn<FloatType>{strings_column});
  }
  // non-integral types throw an exception
  template <typename T, std::enable_if_t<not std::is_floating_point<T>::value>* = nullptr>
  void operator()(column_device_view const&, mutable_column_view&, rmm::cuda_stream_view) const
  {
    CUDF_FAIL("Output for to_floats must be a float type.");
  }
};

}  // namespace

// This will convert a strings column into any float column type.
std::unique_ptr<column> to_floats(strings_column_view const& strings,
                                  data_type output_type,
                                  rmm::cuda_stream_view stream,
                                  rmm::mr::device_memory_resource* mr)
{
  size_type strings_count = strings.size();
  if (strings_count == 0) return make_numeric_column(output_type, 0);
  auto strings_column = column_device_view::create(strings.parent(), stream);
  auto d_strings      = *strings_column;
  // create float output column copying the strings null-mask
  auto results      = make_numeric_column(output_type,
                                     strings_count,
                                     cudf::detail::copy_bitmask(strings.parent(), stream, mr),
                                     strings.null_count(),
                                     stream,
                                     mr);
  auto results_view = results->mutable_view();
  // fill output column with floats
  type_dispatcher(output_type, dispatch_to_floats_fn{}, d_strings, results_view, stream);
  results->set_null_count(strings.null_count());
  return results;
}

}  // namespace detail

// external API

std::unique_ptr<column> to_floats(strings_column_view const& strings,
                                  data_type output_type,
                                  rmm::mr::device_memory_resource* mr)
{
  CUDF_FUNC_RANGE();
  return detail::to_floats(strings, output_type, rmm::cuda_stream_default, mr);
}

namespace detail {
namespace {
/**
 * @brief Code logic for converting float value into a string.
 *
 * The floating point components are dissected and used to fill an
 * existing output char array.
 */
struct ftos_converter {
  // significant digits is independent of scientific notation range
  // digits more than this may require using long values instead of ints
  static constexpr unsigned int significant_digits = 10;
  // maximum power-of-10 that will fit in 32-bits
  static constexpr unsigned int nine_digits = 1000000000;  // 1x10^9
  // Range of numbers here is for normalizing the value.
  // If the value is above or below the following limits, the output is converted to
  // scientific notation in order to show (at most) the number of significant digits.
  static constexpr double upper_limit = 1000000000;  // max is 1x10^9
  static constexpr double lower_limit = 0.0001;      // printf uses scientific notation below this
  // Tables for doing normalization: converting to exponent form
  // IEEE double float has maximum exponent of 305 so these should cover everthing
  const double upper10[9]  = {10, 100, 10000, 1e8, 1e16, 1e32, 1e64, 1e128, 1e256};
  const double lower10[9]  = {.1, .01, .0001, 1e-8, 1e-16, 1e-32, 1e-64, 1e-128, 1e-256};
  const double blower10[9] = {1.0, .1, .001, 1e-7, 1e-15, 1e-31, 1e-63, 1e-127, 1e-255};

  // utility for quickly converting known integer range to character array
  __device__ char* int2str(int value, char* output)
  {
    if (value == 0) {
      *output++ = '0';
      return output;
    }
    char buffer[significant_digits];  // should be big-enough for significant digits
    char* ptr = buffer;
    while (value > 0) {
      *ptr++ = (char)('0' + (value % 10));
      value /= 10;
    }
    while (ptr != buffer) *output++ = *--ptr;  // 54321 -> 12345
    return output;
  }

  /**
   * @brief Dissect a float value into integer, decimal, and exponent components.
   *
   * @return The number of decimal places.
   */
  __device__ int dissect_value(double value,
                               unsigned int& integer,
                               unsigned int& decimal,
                               int& exp10)
  {
    int decimal_places = significant_digits - 1;
    // normalize step puts value between lower-limit and upper-limit
    // by adjusting the exponent up or down
    exp10 = 0;
    if (value > upper_limit) {
      int fx = 256;
      for (int idx = 8; idx >= 0; --idx) {
        if (value >= upper10[idx]) {
          value *= lower10[idx];
          exp10 += fx;
        }
        fx = fx >> 1;
      }
    } else if ((value > 0.0) && (value < lower_limit)) {
      int fx = 256;
      for (int idx = 8; idx >= 0; --idx) {
        if (value < blower10[idx]) {
          value *= upper10[idx];
          exp10 -= fx;
        }
        fx = fx >> 1;
      }
    }
    //
    unsigned int max_digits = nine_digits;
    integer                 = (unsigned int)value;
    for (unsigned int i = integer; i >= 10; i /= 10) {
      --decimal_places;
      max_digits /= 10;
    }
    double remainder = (value - (double)integer) * (double)max_digits;
    decimal          = (unsigned int)remainder;
    remainder -= (double)decimal;
    decimal += (unsigned int)(2.0 * remainder);
    if (decimal >= max_digits) {
      decimal = 0;
      ++integer;
      if (exp10 && (integer >= 10)) {
        ++exp10;
        integer = 1;
      }
    }
    //
    while ((decimal % 10) == 0 && (decimal_places > 0)) {
      decimal /= 10;
      --decimal_places;
    }
    return decimal_places;
  }

  /**
   * @brief Main kernel method for converting float value to char output array.
   *
   * Output need not be more than (significant_digits + 7) bytes:
   * 7 = 1 sign, 1 decimal point, 1 exponent ('e'), 1 exponent-sign, 3 digits for exponent
   *
   * @param value Float value to convert.
   * @param output Memory to write output characters.
   * @return Number of bytes written.
   */
  __device__ int float_to_string(double value, char* output)
  {
    // check for valid value
    if (std::isnan(value)) {
      memcpy(output, "NaN", 3);
      return 3;
    }
    bool bneg = false;
    if (signbit(value)) {  // handles -0.0 too
      value = -value;
      bneg  = true;
    }
    if (std::isinf(value)) {
      if (bneg)
        memcpy(output, "-Inf", 4);
      else
        memcpy(output, "Inf", 3);
      return bneg ? 4 : 3;
    }

    // dissect value into components
    unsigned int integer = 0, decimal = 0;
    int exp10          = 0;
    int decimal_places = dissect_value(value, integer, decimal, exp10);
    //
    // now build the string from the
    // components: sign, integer, decimal, exp10, decimal_places
    //
    // sign
    char* ptr = output;
    if (bneg) *ptr++ = '-';
    // integer
    ptr = int2str(integer, ptr);
    // decimal
    *ptr++ = '.';
    if (decimal_places) {
      char buffer[10];
      char* pb = buffer;
      while (decimal_places--) {
        *pb++ = (char)('0' + (decimal % 10));
        decimal /= 10;
      }
      while (pb != buffer)  // reverses the digits
        *ptr++ = *--pb;     // e.g. 54321 -> 12345
    } else
      *ptr++ = '0';  // always include at least .0
    // exponent
    if (exp10) {
      *ptr++ = 'e';
      if (exp10 < 0) {
        *ptr++ = '-';
        exp10  = -exp10;
      } else
        *ptr++ = '+';
      if (exp10 < 10) *ptr++ = '0';  // extra zero-pad
      ptr = int2str(exp10, ptr);
    }
    // done
    return (int)(ptr - output);  // number of bytes written
  }

  /**
   * @brief Compute how man bytes are needed to hold the output string.
   *
   * @param value Float value to convert.
   * @return Number of bytes required.
   */
  __device__ int compute_ftos_size(double value)
  {
    if (std::isnan(value)) return 3;  // NaN
    bool bneg = false;
    if (signbit(value)) {  // handles -0.0 too
      value = -value;
      bneg  = true;
    }
    if (std::isinf(value)) return 3 + (int)bneg;  // Inf

    // dissect float into parts
    unsigned int integer = 0, decimal = 0;
    int exp10          = 0;
    int decimal_places = dissect_value(value, integer, decimal, exp10);
    // now count up the components
    // sign
    int count = (int)bneg;
    // integer
    count += (int)(integer == 0);
    while (integer > 0) {
      integer /= 10;
      ++count;
    }  // log10(integer)
    // decimal
    ++count;  // decimal point
    if (decimal_places)
      count += decimal_places;
    else
      ++count;  // always include .0
    // exponent
    if (exp10) {
      count += 2;  // 'e±'
      if (exp10 < 0) exp10 = -exp10;
      count += (int)(exp10 < 10);  // padding
      while (exp10 > 0) {
        exp10 /= 10;
        ++count;
      }  // log10(exp10)
    }
    return count;
  }
};

template <typename FloatType>
struct float_to_string_size_fn {
  column_device_view d_column;

  __device__ size_type operator()(size_type idx)
  {
    if (d_column.is_null(idx)) return 0;
    FloatType value = d_column.element<FloatType>(idx);
    ftos_converter fts;
    return static_cast<size_type>(fts.compute_ftos_size(static_cast<double>(value)));
  }
};

template <typename FloatType>
struct float_to_string_fn {
  const column_device_view d_column;
  const int32_t* d_offsets;
  char* d_chars;

  __device__ void operator()(size_type idx)
  {
    if (d_column.is_null(idx)) return;
    FloatType value = d_column.element<FloatType>(idx);
    ftos_converter fts;
    fts.float_to_string(static_cast<double>(value), d_chars + d_offsets[idx]);
  }
};

/**
 * @brief This dispatch method is for converting floats into strings.
 *
 * The template function declaration ensures only float types are allowed.
 */
struct dispatch_from_floats_fn {
  template <typename FloatType,
            std::enable_if_t<std::is_floating_point<FloatType>::value>* = nullptr>
  std::unique_ptr<column> operator()(column_view const& floats,
                                     rmm::cuda_stream_view stream,
                                     rmm::mr::device_memory_resource* mr) const
  {
    size_type strings_count = floats.size();
    auto column             = column_device_view::create(floats, stream);
    auto d_column           = *column;

    // copy the null mask
    rmm::device_buffer null_mask = cudf::detail::copy_bitmask(floats, stream, mr);
    // build offsets column
    auto offsets_transformer_itr = thrust::make_transform_iterator(
      thrust::make_counting_iterator<int32_t>(0), float_to_string_size_fn<FloatType>{d_column});
    auto offsets_column = detail::make_offsets_child_column(
      offsets_transformer_itr, offsets_transformer_itr + strings_count, stream, mr);
    auto offsets_view = offsets_column->view();
    auto d_offsets    = offsets_view.template data<int32_t>();

    // build chars column
    size_type bytes = thrust::device_pointer_cast(d_offsets)[strings_count];
    auto chars_column =
      detail::create_chars_child_column(strings_count, floats.null_count(), bytes, stream, mr);
    auto chars_view = chars_column->mutable_view();
    auto d_chars    = chars_view.template data<char>();
    thrust::for_each_n(rmm::exec_policy(stream),
                       thrust::make_counting_iterator<size_type>(0),
                       strings_count,
                       float_to_string_fn<FloatType>{d_column, d_offsets, d_chars});
    //
    return make_strings_column(strings_count,
                               std::move(offsets_column),
                               std::move(chars_column),
                               floats.null_count(),
                               std::move(null_mask),
                               stream,
                               mr);
  }

  // non-float types throw an exception
  template <typename T, std::enable_if_t<not std::is_floating_point<T>::value>* = nullptr>
  std::unique_ptr<column> operator()(column_view const&,
                                     rmm::cuda_stream_view,
                                     rmm::mr::device_memory_resource*) const
  {
    CUDF_FAIL("Values for from_floats function must be a float type.");
  }
};

}  // namespace

// This will convert all float column types into a strings column.
std::unique_ptr<column> from_floats(column_view const& floats,
                                    rmm::cuda_stream_view stream,
                                    rmm::mr::device_memory_resource* mr)
{
  size_type strings_count = floats.size();
  if (strings_count == 0) return detail::make_empty_strings_column(stream, mr);

  return type_dispatcher(floats.type(), dispatch_from_floats_fn{}, floats, stream, mr);
}

}  // namespace detail

// external API

std::unique_ptr<column> from_floats(column_view const& floats, rmm::mr::device_memory_resource* mr)
{
  CUDF_FUNC_RANGE();
  return detail::from_floats(floats, rmm::cuda_stream_default, mr);
}

}  // namespace strings
}  // namespace cudf
