#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2021-2024, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <cudf/column/column_device_view.cuh>
#include <cudf/column/column_factories.hpp>
#include <cudf/detail/iterator.cuh>
#include <cudf/detail/null_mask.hpp>
#include <cudf/detail/nvtx/ranges.hpp>
#include <cudf/strings/convert/convert_fixed_point.hpp>
#include <cudf/strings/detail/convert/fixed_point.cuh>
#include <cudf/strings/detail/convert/fixed_point_to_string.cuh>
#include <cudf/strings/detail/converters.hpp>
#include <cudf/strings/detail/strings_children.cuh>
#include <cudf/strings/string_view.cuh>
#include <cudf/strings/strings_column_view.hpp>
#include <cudf/utilities/default_stream.hpp>
#include <cudf/utilities/type_dispatcher.hpp>

#include <rmm/cuda_stream_view.hpp>
#include <rmm/exec_policy.hpp>
#include <rmm/resource_ref.hpp>

#include <cuda/std/climits>
#include <cuda/std/limits>
#include <cuda/std/type_traits>
#include <thrust/execution_policy.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/transform.h>

namespace cudf {
namespace strings {
namespace detail {
namespace {

/**
 * @brief Converts strings into an integers and records decimal places.
 *
 * The conversion uses the provided scale to build the resulting
 * integer. This can prevent overflow for strings with many digits.
 */
template <typename DecimalType>
struct string_to_decimal_fn {
  column_device_view const d_strings;
  int32_t const scale;

  string_to_decimal_fn(column_device_view const& d_strings, int32_t scale)
    : d_strings(d_strings), scale(scale)
  {
  }

  __device__ DecimalType operator()(size_type idx) const
  {
    if (d_strings.is_null(idx)) { return 0; }
    auto const d_str = d_strings.element<string_view>(idx);
    if (d_str.empty()) { return 0; }

    auto iter           = d_str.data();
    auto const iter_end = d_str.data() + d_str.size_bytes();

    return parse_decimal<DecimalType>(iter, iter_end, scale);
  }
};

/**
 * @brief This only checks the string format for valid decimal characters.
 *
 * This follows closely the logic above but just ensures there are valid
 * characters for conversion and the integer component does not overflow.
 */
template <typename DecimalType>
struct string_to_decimal_check_fn {
  column_device_view const d_strings;
  int32_t const scale;

  string_to_decimal_check_fn(column_device_view const& d_strings, int32_t scale)
    : d_strings{d_strings}, scale{scale}
  {
  }

  __device__ bool operator()(size_type idx) const
  {
    if (d_strings.is_null(idx)) { return false; }
    auto const d_str = d_strings.element<string_view>(idx);
    if (d_str.empty()) { return false; }

    auto iter = d_str.data() + static_cast<int>((d_str.data()[0] == '-' || d_str.data()[0] == '+'));

    auto const iter_end = d_str.data() + d_str.size_bytes();

    using UnsignedDecimalType = cuda::std::make_unsigned_t<DecimalType>;
    auto [value, exp_offset]  = parse_integer<UnsignedDecimalType>(iter, iter_end);

    // only exponent notation is expected here
    if ((iter < iter_end) && (*iter != 'e' && *iter != 'E')) { return false; }
    ++iter;

    int32_t exp_ten = 0;  // check exponent overflow
    if (iter < iter_end) {
      auto exp_result = parse_exponent<true>(iter, iter_end);
      if (!exp_result) { return false; }
      exp_ten = exp_result.value();
    }
    exp_ten += exp_offset;

    // finally, check for overflow based on the exp_ten and scale values
    return (exp_ten < scale) or
           value <= static_cast<UnsignedDecimalType>(
                      cuda::std::numeric_limits<DecimalType>::max() /
                      static_cast<DecimalType>(exp10(static_cast<double>(exp_ten - scale))));
  }
};

/**
 * @brief The dispatch function for converting strings column to fixed-point column.
 */
struct dispatch_to_fixed_point_fn {
  template <typename T, std::enable_if_t<cudf::is_fixed_point<T>()>* = nullptr>
  std::unique_ptr<column> operator()(strings_column_view const& input,
                                     data_type output_type,
                                     rmm::cuda_stream_view stream,
                                     rmm::device_async_resource_ref mr) const
  {
    using DecimalType = device_storage_type_t<T>;

    auto const d_column = column_device_view::create(input.parent(), stream);

    // create output column
    auto results   = make_fixed_point_column(output_type,
                                           input.size(),
                                           cudf::detail::copy_bitmask(input.parent(), stream, mr),
                                           input.null_count(),
                                           stream,
                                           mr);
    auto d_results = results->mutable_view().data<DecimalType>();

    // convert strings into decimal values
    thrust::transform(rmm::exec_policy(stream),
                      thrust::make_counting_iterator<size_type>(0),
                      thrust::make_counting_iterator<size_type>(input.size()),
                      d_results,
                      string_to_decimal_fn<DecimalType>{*d_column, output_type.scale()});
    results->set_null_count(input.null_count());
    return results;
  }

  template <typename T, std::enable_if_t<not cudf::is_fixed_point<T>()>* = nullptr>
  std::unique_ptr<column> operator()(strings_column_view const&,
                                     data_type,
                                     rmm::cuda_stream_view,
                                     rmm::device_async_resource_ref) const
  {
    CUDF_FAIL("Output for to_fixed_point must be a decimal type.");
  }
};

}  // namespace

// This will convert a strings column into any integer column type.
std::unique_ptr<column> to_fixed_point(strings_column_view const& input,
                                       data_type output_type,
                                       rmm::cuda_stream_view stream,
                                       rmm::device_async_resource_ref mr)
{
  if (input.is_empty()) return make_empty_column(output_type);
  return type_dispatcher(output_type, dispatch_to_fixed_point_fn{}, input, output_type, stream, mr);
}

}  // namespace detail

// external API
std::unique_ptr<column> to_fixed_point(strings_column_view const& input,
                                       data_type output_type,
                                       rmm::cuda_stream_view stream,
                                       rmm::device_async_resource_ref mr)
{
  CUDF_FUNC_RANGE();
  return detail::to_fixed_point(input, output_type, stream, mr);
}

namespace detail {
namespace {
template <typename DecimalType>
struct from_fixed_point_fn {
  column_device_view d_decimals;
  size_type* d_sizes{};
  char* d_chars{};
  cudf::detail::input_offsetalator d_offsets;

  /**
   * @brief Converts a decimal element into a string.
   *
   * The value is converted into base-10 digits [0-9]
   * plus the decimal point and a negative sign prefix.
   */
  __device__ void fixed_point_element_to_string(size_type idx)
  {
    auto const value = d_decimals.element<DecimalType>(idx);
    auto const scale = d_decimals.type().scale();
    char* d_buffer   = d_chars + d_offsets[idx];

    fixed_point_to_string(value, scale, d_buffer);
  }

  __device__ void operator()(size_type idx)
  {
    if (d_decimals.is_null(idx)) {
      if (d_chars == nullptr) { d_sizes[idx] = 0; }
      return;
    }
    if (d_chars != nullptr) {
      fixed_point_element_to_string(idx);
    } else {
      d_sizes[idx] =
        fixed_point_string_size(d_decimals.element<DecimalType>(idx), d_decimals.type().scale());
    }
  }
};

/**
 * @brief The dispatcher functor for converting fixed-point values into strings.
 */
struct dispatch_from_fixed_point_fn {
  template <typename T, std::enable_if_t<cudf::is_fixed_point<T>()>* = nullptr>
  std::unique_ptr<column> operator()(column_view const& input,
                                     rmm::cuda_stream_view stream,
                                     rmm::device_async_resource_ref mr) const
  {
    using DecimalType = device_storage_type_t<T>;  // underlying value type

    auto const d_column = column_device_view::create(input, stream);

    auto [offsets, chars] =
      make_strings_children(from_fixed_point_fn<DecimalType>{*d_column}, input.size(), stream, mr);

    return make_strings_column(input.size(),
                               std::move(offsets),
                               chars.release(),
                               input.null_count(),
                               cudf::detail::copy_bitmask(input, stream, mr));
  }

  template <typename T, std::enable_if_t<not cudf::is_fixed_point<T>()>* = nullptr>
  std::unique_ptr<column> operator()(column_view const&,
                                     rmm::cuda_stream_view,
                                     rmm::device_async_resource_ref) const
  {
    CUDF_FAIL("Values for from_fixed_point function must be a decimal type.");
  }
};

}  // namespace

std::unique_ptr<column> from_fixed_point(column_view const& input,
                                         rmm::cuda_stream_view stream,
                                         rmm::device_async_resource_ref mr)
{
  if (input.is_empty()) return make_empty_column(type_id::STRING);
  return type_dispatcher(input.type(), dispatch_from_fixed_point_fn{}, input, stream, mr);
}

}  // namespace detail

// external API

std::unique_ptr<column> from_fixed_point(column_view const& input,
                                         rmm::cuda_stream_view stream,
                                         rmm::device_async_resource_ref mr)
{
  CUDF_FUNC_RANGE();
  return detail::from_fixed_point(input, stream, mr);
}

namespace detail {
namespace {

struct dispatch_is_fixed_point_fn {
  template <typename T, std::enable_if_t<cudf::is_fixed_point<T>()>* = nullptr>
  std::unique_ptr<column> operator()(strings_column_view const& input,
                                     data_type decimal_type,
                                     rmm::cuda_stream_view stream,
                                     rmm::device_async_resource_ref mr) const
  {
    using DecimalType = device_storage_type_t<T>;

    auto const d_column = column_device_view::create(input.parent(), stream);

    // create output column
    auto results   = make_numeric_column(data_type{type_id::BOOL8},
                                       input.size(),
                                       cudf::detail::copy_bitmask(input.parent(), stream, mr),
                                       input.null_count(),
                                       stream,
                                       mr);
    auto d_results = results->mutable_view().data<bool>();

    // check strings for valid fixed-point chars
    thrust::transform(rmm::exec_policy(stream),
                      thrust::make_counting_iterator<size_type>(0),
                      thrust::make_counting_iterator<size_type>(input.size()),
                      d_results,
                      string_to_decimal_check_fn<DecimalType>{*d_column, decimal_type.scale()});
    results->set_null_count(input.null_count());
    return results;
  }

  template <typename T, std::enable_if_t<not cudf::is_fixed_point<T>()>* = nullptr>
  std::unique_ptr<column> operator()(strings_column_view const&,
                                     data_type,
                                     rmm::cuda_stream_view,
                                     rmm::device_async_resource_ref) const
  {
    CUDF_FAIL("is_fixed_point is expecting a decimal type");
  }
};

}  // namespace

std::unique_ptr<column> is_fixed_point(strings_column_view const& input,
                                       data_type decimal_type,
                                       rmm::cuda_stream_view stream,
                                       rmm::device_async_resource_ref mr)
{
  if (input.is_empty()) return cudf::make_empty_column(type_id::BOOL8);
  return type_dispatcher(
    decimal_type, dispatch_is_fixed_point_fn{}, input, decimal_type, stream, mr);
}
}  // namespace detail

std::unique_ptr<column> is_fixed_point(strings_column_view const& input,
                                       data_type decimal_type,
                                       rmm::cuda_stream_view stream,
                                       rmm::device_async_resource_ref mr)
{
  CUDF_FUNC_RANGE();
  return detail::is_fixed_point(input, decimal_type, stream, mr);
}

}  // namespace strings
}  // namespace cudf
