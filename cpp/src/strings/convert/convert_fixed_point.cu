#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2021, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <cudf/column/column_device_view.cuh>
#include <cudf/column/column_factories.hpp>
#include <cudf/detail/get_value.cuh>
#include <cudf/detail/null_mask.hpp>
#include <cudf/detail/nvtx/ranges.hpp>
#include <cudf/detail/utilities/integer_utils.hpp>
#include <cudf/strings/convert/convert_fixed_point.hpp>
#include <cudf/strings/detail/converters.hpp>
#include <cudf/strings/detail/utilities.hpp>
#include <cudf/strings/string_view.cuh>
#include <cudf/strings/strings_column_view.hpp>
#include <cudf/utilities/traits.hpp>
#include <cudf/utilities/type_dispatcher.hpp>

#include <strings/convert/utilities.cuh>
#include <strings/utilities.cuh>

#include <rmm/cuda_stream_view.hpp>
#include <rmm/device_uvector.hpp>
#include <rmm/exec_policy.hpp>

#include <thrust/extrema.h>
#include <thrust/iterator/counting_iterator.h>

namespace cudf {
namespace strings {
namespace detail {
namespace {
/**
 * @brief Converts strings into an integers and records decimal point.
 *
 * Used by the dispatch method to convert to different fixed-point types.
 */
template <typename DecimalType>
struct string_to_decimal_fn {
  column_device_view const d_strings;
  DecimalType* values;
  int32_t* scales;

  __device__ void operator()(size_type idx)
  {
    values[idx] = DecimalType{0};
    scales[idx] = numeric::scale_type{0};
    if (d_strings.is_null(idx)) return;
    auto const d_str = d_strings.element<string_view>(idx);
    if (d_str.empty()) return;
    auto const sign = [&] {
      auto const first = d_str.data();
      if (*first == '-') return -1;
      if (*first == '+') return 1;
      return 0;
    }();
    auto iter = d_str.begin() + (sign != 0);

    int64_t value      = 0;
    bool decimal_found = false;
    while (iter != d_str.end()) {
      auto const chr = *iter++;
      if (chr >= '0' && chr <= '9') {
        value = (value * 10) + static_cast<int64_t>(chr - '0');
        scales[idx] -= static_cast<int32_t>(decimal_found);
      } else if (chr == '.')
        decimal_found = true;
      else
        break;
    }
    values[idx] = static_cast<DecimalType>(value * (sign == 0 ? 1 : sign));
  }
};

template <typename DecimalType>
struct rescale_decimals_fn {
  column_device_view const d_strings;
  int32_t const max_scale;
  int32_t const* scales;
  DecimalType* values;

  __device__ void operator()(size_type idx)
  {
    if (d_strings.is_null(idx)) return;
    numeric::scaled_integer<DecimalType> si{values[idx], numeric::scale_type{scales[idx]}};
    numeric::fixed_point<DecimalType, numeric::Radix::BASE_10> fp{si};
    values[idx] = fp.rescaled(numeric::scale_type{max_scale}).value();
  }
};

/**
 * @brief The dispatch functions for converting strings.
 *
 * The output_column is expected to be one of the integer types only.
 */
struct dispatch_to_fixed_point_fn {
  template <typename T, std::enable_if_t<cudf::is_fixed_point<T>()>* = nullptr>
  std::unique_ptr<column> operator()(strings_column_view const& input,
                                     rmm::cuda_stream_view stream,
                                     rmm::mr::device_memory_resource* mr) const
  {
    using DecimalType = device_storage_type_t<T>;
    rmm::device_uvector<int32_t> d_scales(input.size(), stream);
    rmm::device_uvector<DecimalType> d_values(input.size(), stream, mr);
    auto d_column = column_device_view::create(input.parent(), stream);
    thrust::for_each_n(
      rmm::exec_policy(stream),
      thrust::make_counting_iterator<size_type>(0),
      input.size(),
      string_to_decimal_fn<DecimalType>{*d_column, d_values.data(), d_scales.data()});
    // find the maximum scale size -- min is used since all scale values will be <= 0
    auto const min_elem =
      thrust::min_element(rmm::exec_policy(stream), d_scales.begin(), d_scales.end());
    auto const scale = d_scales.element(thrust::distance(d_scales.begin(), min_elem), stream);
    // re-scale all the values to the max scale
    thrust::for_each_n(
      rmm::exec_policy(stream),
      thrust::make_counting_iterator<size_type>(0),
      input.size(),
      rescale_decimals_fn<DecimalType>{*d_column, scale, d_scales.data(), d_values.data()});
    // build output column
    return std::make_unique<column>(data_type{type_to_id<T>(), scale},
                                    input.size(),
                                    d_values.release(),
                                    cudf::detail::copy_bitmask(input.parent(), stream, mr),
                                    input.null_count());
  }
  // non-integral types throw an exception
  template <typename T, std::enable_if_t<not cudf::is_fixed_point<T>()>* = nullptr>
  std::unique_ptr<column> operator()(strings_column_view const&,
                                     rmm::cuda_stream_view,
                                     rmm::mr::device_memory_resource*) const
  {
    CUDF_FAIL("Output for to_fixed_point must be an fixed-point type.");
  }
};

}  // namespace

// This will convert a strings column into any integer column type.
std::unique_ptr<column> to_fixed_point(strings_column_view const& input,
                                       data_type output_type,
                                       rmm::cuda_stream_view stream,
                                       rmm::mr::device_memory_resource* mr)
{
  if (input.is_empty()) return make_empty_column(output_type);
  return type_dispatcher(output_type, dispatch_to_fixed_point_fn{}, input, stream, mr);
}

}  // namespace detail

// external API
std::unique_ptr<column> to_fixed_point(strings_column_view const& strings,
                                       data_type output_type,
                                       rmm::mr::device_memory_resource* mr)
{
  CUDF_FUNC_RANGE();
  return detail::to_fixed_point(strings, output_type, rmm::cuda_stream_default, mr);
}

namespace detail {
namespace {
/**
 * @brief Calculate the size of the each string required for
 * converting each value in base-10 format.
 */
template <typename DecimalType>
struct decimal_to_string_size_fn {
  column_device_view const d_column;

  __device__ int32_t operator()(size_type idx) const
  {
    if (d_column.is_null(idx)) return 0;
    auto const value = d_column.element<DecimalType>(idx);
    auto const scale = d_column.type().scale();

    if (scale >= 0) return count_digits(value) + scale;

    auto const abs_value = std::abs(value);
    auto const exp_ten   = static_cast<int32_t>(exp10(static_cast<double>(-scale)));
    auto const num_zeros = std::max(0, (-scale - count_digits(abs_value % exp_ten)));
    return static_cast<int32_t>(value < 0) + count_digits(abs_value / exp_ten) + 1 + num_zeros +
           count_digits(abs_value % exp_ten);
  }
};

/**
 * @brief Convert each value into a string.
 *
 * The value is converted into base-10 using only characters [0-9].
 */
template <typename DecimalType>
struct decimal_to_string_fn {
  column_device_view const d_column;
  int32_t const* d_offsets;
  char* d_chars;

  __device__ void operator()(size_type idx)
  {
    if (d_column.is_null(idx)) return;
    auto const value = d_column.element<DecimalType>(idx);
    auto const scale = d_column.type().scale();
    char* d_buffer   = d_chars + d_offsets[idx];
    if (scale >= 0) {
      integer_to_string(value, d_buffer);
      d_buffer += count_digits(value);
      // add zeros
      thrust::generate_n(thrust::seq, d_buffer, scale, []() { return '0'; });
      return;
    }

    // scale < 0
    // write format:   [-]integer.fraction
    // where integer  = abs(value) / (10 ^ abs(scale))
    //       fraction = abs(value) % (10 ^ abs(scale))
    auto const abs_value = std::abs(value);
    if (value < 0) *d_buffer++ = '-';  // add sign
    auto const exp_ten   = static_cast<int32_t>(exp10(static_cast<double>(-scale)));
    auto const num_zeros = std::max(0, (-scale - count_digits(abs_value % exp_ten)));
    // add the integer part
    integer_to_string(abs_value / exp_ten, d_buffer);
    d_buffer += count_digits(abs_value / exp_ten);
    *d_buffer++ = '.';  // add decimal point
    // add zeros
    thrust::generate_n(thrust::seq, d_buffer, num_zeros, []() { return '0'; });
    d_buffer += num_zeros;
    // add the fractional part
    integer_to_string(abs_value % exp_ten, d_buffer);
  }
};

/**
 * @brief The dispatcher functor for converting fixed-point values into strings.
 */
struct dispatch_from_fixed_point_fn {
  template <typename T, std::enable_if_t<cudf::is_fixed_point<T>()>* = nullptr>
  std::unique_ptr<column> operator()(column_view const& input,
                                     rmm::cuda_stream_view stream,
                                     rmm::mr::device_memory_resource* mr) const
  {
    using DecimalType = device_storage_type_t<T>;  // underlying value type
    auto d_column     = column_device_view::create(input, stream);

    // build offsets column
    // !!!!!!!!!!!!!!!!!!!! use cudf::detail::make-something-iterator
    auto offsets_transformer_itr =
      thrust::make_transform_iterator(thrust::make_counting_iterator<int32_t>(0),
                                      decimal_to_string_size_fn<DecimalType>{*d_column});
    // !!!!!!!!!!!!!!!!!!!!
    auto offsets_column = detail::make_offsets_child_column(
      offsets_transformer_itr, offsets_transformer_itr + input.size(), stream, mr);
    auto d_offsets = offsets_column->view().template data<int32_t>();

    // build chars column
    auto const bytes =
      cudf::detail::get_value<int32_t>(offsets_column->view(), input.size(), stream);
    auto chars_column =
      detail::create_chars_child_column(input.size(), input.null_count(), bytes, stream, mr);
    auto chars_view = chars_column->mutable_view();
    auto d_chars    = chars_view.template data<char>();
    thrust::for_each_n(rmm::exec_policy(stream),
                       thrust::make_counting_iterator<size_type>(0),
                       input.size(),
                       decimal_to_string_fn<DecimalType>{*d_column, d_offsets, d_chars});

    return make_strings_column(input.size(),
                               std::move(offsets_column),
                               std::move(chars_column),
                               input.null_count(),
                               cudf::detail::copy_bitmask(input, stream, mr),
                               stream,
                               mr);
  }

  // non-integral types throw an exception
  template <typename T, std::enable_if_t<not cudf::is_fixed_point<T>()>* = nullptr>
  std::unique_ptr<column> operator()(column_view const&,
                                     rmm::cuda_stream_view,
                                     rmm::mr::device_memory_resource*) const
  {
    CUDF_FAIL("Values for from_fixed_point function must be an fixed-point type.");
  }
};

}  // namespace

// This will convert all integer column types into a strings column.
std::unique_ptr<column> from_fixed_point(column_view const& integers,
                                         rmm::cuda_stream_view stream,
                                         rmm::mr::device_memory_resource* mr)
{
  size_type strings_count = integers.size();
  if (strings_count == 0) return detail::make_empty_strings_column(stream, mr);

  return type_dispatcher(integers.type(), dispatch_from_fixed_point_fn{}, integers, stream, mr);
}

}  // namespace detail

// external API

std::unique_ptr<column> from_fixed_point(column_view const& integers,
                                         rmm::mr::device_memory_resource* mr)
{
  CUDF_FUNC_RANGE();
  return detail::from_fixed_point(integers, rmm::cuda_stream_default, mr);
}

}  // namespace strings
}  // namespace cudf
