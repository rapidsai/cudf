#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2021, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <cudf/column/column_device_view.cuh>
#include <cudf/column/column_factories.hpp>
#include <cudf/detail/get_value.cuh>
#include <cudf/detail/iterator.cuh>
#include <cudf/detail/null_mask.hpp>
#include <cudf/detail/nvtx/ranges.hpp>
#include <cudf/strings/convert/convert_fixed_point.hpp>
#include <cudf/strings/detail/converters.hpp>
#include <cudf/strings/detail/utilities.cuh>
#include <cudf/strings/detail/utilities.hpp>
#include <cudf/strings/string_view.cuh>
#include <cudf/strings/strings_column_view.hpp>
#include <cudf/utilities/type_dispatcher.hpp>

#include <strings/convert/utilities.cuh>

#include <rmm/cuda_stream_view.hpp>
#include <rmm/exec_policy.hpp>

#include <thrust/optional.h>
#include <thrust/transform.h>

namespace cudf {
namespace strings {
namespace detail {
namespace {

struct string_to_decimal_base {
  /**
   * @brief Return the integer component of a decimal string.
   *
   * This is reads everything up to the exponent 'e' notation.
   * The return includes the integer digits and any exponent offset.
   *
   * @param[in,out] iter Start of characters to parse
   * @param[in] end End of characters to parse
   * @return Integer component and exponent offset.
   */
  __device__ thrust::pair<uint64_t, int32_t> parse_integer(char const*& iter,
                                                           char const* iter_end) const
  {
    // highest value where another decimal digit cannot be appended without an overflow;
    // this preserves the most digits when scaling the final result
    constexpr uint64_t decimal_max = (std::numeric_limits<uint64_t>::max() - 9L) / 10L;

    uint64_t value     = 0;  // for checking overflow
    int32_t exp_offset = 0;
    bool decimal_found = false;

    while (iter < iter_end) {
      auto const ch = *iter++;
      if (ch == '.' && !decimal_found) {
        decimal_found = true;
        continue;
      }
      if (ch < '0' || ch > '9') {
        --iter;
        break;
      }
      if (value > decimal_max) {
        exp_offset += static_cast<int32_t>(!decimal_found);
      } else {
        value = (value * 10) + static_cast<uint64_t>(ch - '0');
        exp_offset -= static_cast<int32_t>(decimal_found);
      }
    }
    return {value, exp_offset};
  }

  /**
   * @brief Return the exponent of a decimal string.
   *
   * This should only be called after the exponent 'e' notation was detected.
   * The return is the exponent (base-10) integer and can only be
   * invalid if `check_only == true` and invalid characters are found or the
   * exponent overflows an int32.
   *
   * @tparam check_only Set to true to verify the characters are valid and the
   *         exponent value in the decimal string does not overflow int32
   * @param[in,out] iter Start of characters to parse
   *                     (points to the character after the 'E' or 'e')
   * @param[in] end End of characters to parse
   * @return Integer value of the exponent
   */
  template <bool check_only = false>
  __device__ thrust::optional<int32_t> parse_exponent(char const* iter, char const* iter_end) const
  {
    constexpr uint32_t exponent_max = static_cast<uint32_t>(std::numeric_limits<int32_t>::max());

    // get optional exponent sign
    int32_t const exp_sign = [&iter] {
      auto const ch = *iter;
      if (ch != '-' && ch != '+') { return 1; }
      ++iter;
      return (ch == '-' ? -1 : 1);
    }();

    // parse exponent integer
    int32_t exp_ten = 0;
    while (iter < iter_end) {
      auto const ch = *iter++;
      if (ch < '0' || ch > '9') {
        if (check_only) { return thrust::nullopt; }
        break;
      }

      uint32_t exp_check = static_cast<uint32_t>(exp_ten * 10) + static_cast<uint32_t>(ch - '0');
      if (check_only && (exp_check > exponent_max)) { return thrust::nullopt; }  // check overflow
      exp_ten = static_cast<int32_t>(exp_check);
    }

    return exp_ten * exp_sign;
  }
};

/**
 * @brief Converts strings into an integers and records decimal places.
 *
 * The conversion uses the provided scale to build the resulting
 * integer. This can prevent overflow for strings with many digits.
 */
template <typename DecimalType>
struct string_to_decimal_fn : string_to_decimal_base {
  column_device_view const d_strings;
  int32_t const scale;

  string_to_decimal_fn(column_device_view const& d_strings, int32_t scale)
    : d_strings(d_strings), scale(scale)
  {
  }

  __device__ DecimalType operator()(size_type idx) const
  {
    if (d_strings.is_null(idx)) { return 0; }
    auto const d_str = d_strings.element<string_view>(idx);
    if (d_str.empty()) { return 0; }

    auto const sign = [&] {
      if (d_str.data()[0] == '-') { return -1; }
      if (d_str.data()[0] == '+') { return 1; }
      return 0;
    }();
    auto iter = d_str.data() + (sign != 0);

    auto const iter_end = d_str.data() + d_str.size_bytes();

    auto [value, exp_offset] = parse_integer(iter, iter_end);
    if (value == 0) { return DecimalType{0}; }

    // check for exponent
    int32_t exp_ten = 0;
    if ((iter < iter_end) && (*iter == 'e' || *iter == 'E')) {
      ++iter;
      if (iter < iter_end) { exp_ten = parse_exponent<false>(iter, iter_end).value(); }
    }
    exp_ten += exp_offset;

    // shift the output value based on the exp_ten and the scale values
    if (exp_ten < scale) {
      value = value / static_cast<uint64_t>(exp10(static_cast<double>(scale - exp_ten)));
    } else {
      value = value * static_cast<uint64_t>(exp10(static_cast<double>(exp_ten - scale)));
    }

    return static_cast<DecimalType>(value) * (sign == 0 ? 1 : sign);
  }
};

/**
 * @brief This only checks the string format for valid decimal characters.
 *
 * This follows closely the logic above but just ensures there are valid
 * characters for conversion and the integer component does not overflow.
 */
template <typename DecimalType>
struct string_to_decimal_check_fn : string_to_decimal_base {
  column_device_view const d_strings;
  int32_t const scale;

  string_to_decimal_check_fn(column_device_view const& d_strings, int32_t scale)
    : d_strings(d_strings), scale(scale)
  {
  }

  __device__ bool operator()(size_type idx) const
  {
    if (d_strings.is_null(idx)) { return false; }
    auto const d_str = d_strings.element<string_view>(idx);
    if (d_str.empty()) { return false; }

    auto iter = d_str.data() + static_cast<int>((d_str.data()[0] == '-' || d_str.data()[0] == '+'));

    auto const iter_end = d_str.data() + d_str.size_bytes();

    auto [value, exp_offset] = parse_integer(iter, iter_end);

    // only exponent notation is expected here
    if ((iter < iter_end) && (*iter != 'e' && *iter != 'E')) { return false; }
    ++iter;

    int32_t exp_ten = 0;  // check exponent overflow
    if (iter < iter_end) {
      auto exp_result = parse_exponent<true>(iter, iter_end);
      if (!exp_result) { return false; }
      exp_ten = exp_result.value();
    }
    exp_ten += exp_offset;

    // finally, check for overflow based on the exp_ten and scale values
    return (exp_ten < scale)
             ? true
             : value <= static_cast<uint64_t>(
                          std::numeric_limits<DecimalType>::max() /
                          static_cast<DecimalType>(exp10(static_cast<double>(exp_ten - scale))));
  }
};

/**
 * @brief The dispatch function for converting strings column to fixed-point column.
 */
struct dispatch_to_fixed_point_fn {
  template <typename T, std::enable_if_t<cudf::is_fixed_point<T>()>* = nullptr>
  std::unique_ptr<column> operator()(strings_column_view const& input,
                                     data_type output_type,
                                     rmm::cuda_stream_view stream,
                                     rmm::mr::device_memory_resource* mr) const
  {
    using DecimalType = device_storage_type_t<T>;

    auto const d_column = column_device_view::create(input.parent(), stream);

    // create output column
    auto results   = make_fixed_point_column(output_type,
                                           input.size(),
                                           cudf::detail::copy_bitmask(input.parent(), stream, mr),
                                           input.null_count(),
                                           stream,
                                           mr);
    auto d_results = results->mutable_view().data<DecimalType>();

    // convert strings into decimal values
    thrust::transform(rmm::exec_policy(stream),
                      thrust::make_counting_iterator<size_type>(0),
                      thrust::make_counting_iterator<size_type>(input.size()),
                      d_results,
                      string_to_decimal_fn<DecimalType>{*d_column, output_type.scale()});
    results->set_null_count(input.null_count());
    return results;
  }

  template <typename T, std::enable_if_t<not cudf::is_fixed_point<T>()>* = nullptr>
  std::unique_ptr<column> operator()(strings_column_view const&,
                                     data_type,
                                     rmm::cuda_stream_view,
                                     rmm::mr::device_memory_resource*) const
  {
    CUDF_FAIL("Output for to_fixed_point must be a decimal type.");
  }
};

}  // namespace

// This will convert a strings column into any integer column type.
std::unique_ptr<column> to_fixed_point(strings_column_view const& input,
                                       data_type output_type,
                                       rmm::cuda_stream_view stream,
                                       rmm::mr::device_memory_resource* mr)
{
  if (input.is_empty()) return make_empty_column(output_type);
  return type_dispatcher(output_type, dispatch_to_fixed_point_fn{}, input, output_type, stream, mr);
}

}  // namespace detail

// external API
std::unique_ptr<column> to_fixed_point(strings_column_view const& strings,
                                       data_type output_type,
                                       rmm::mr::device_memory_resource* mr)
{
  CUDF_FUNC_RANGE();
  return detail::to_fixed_point(strings, output_type, rmm::cuda_stream_default, mr);
}

namespace detail {
namespace {
/**
 * @brief Calculate the size of the each string required for
 * converting each value in base-10 format.
 *
 * ouput format is [-]integer.fraction
 */
template <typename DecimalType>
struct decimal_to_string_size_fn {
  column_device_view const d_column;

  __device__ int32_t operator()(size_type idx) const
  {
    if (d_column.is_null(idx)) return 0;
    auto const value = d_column.element<DecimalType>(idx);
    auto const scale = d_column.type().scale();

    if (scale >= 0) return count_digits(value) + scale;

    auto const abs_value = std::abs(value);
    auto const exp_ten   = static_cast<int64_t>(exp10(static_cast<double>(-scale)));
    auto const fraction  = count_digits(abs_value % exp_ten);
    auto const num_zeros = std::max(0, (-scale - fraction));
    return static_cast<int32_t>(value < 0) +    // sign if negative
           count_digits(abs_value / exp_ten) +  // integer
           1 +                                  // decimal point
           num_zeros +                          // zeros padding
           fraction;                            // size of fraction
  }
};

/**
 * @brief Convert each value into a string.
 *
 * The value is converted into base-10 digits [0-9]
 * plus the decimal point and a negative sign prefix.
 */
template <typename DecimalType>
struct decimal_to_string_fn {
  column_device_view const d_column;
  int32_t const* d_offsets;
  char* d_chars;

  __device__ void operator()(size_type idx)
  {
    if (d_column.is_null(idx)) return;
    auto const value = d_column.element<DecimalType>(idx);
    auto const scale = d_column.type().scale();
    char* d_buffer   = d_chars + d_offsets[idx];

    if (scale >= 0) {
      d_buffer += integer_to_string(value, d_buffer);
      thrust::generate_n(thrust::seq, d_buffer, scale, []() { return '0'; });  // add zeros
      return;
    }

    // scale < 0
    // write format:   [-]integer.fraction
    // where integer  = abs(value) / (10^abs(scale))
    //       fraction = abs(value) % (10^abs(scale))
    auto const abs_value = std::abs(value);
    if (value < 0) *d_buffer++ = '-';  // add sign
    auto const exp_ten   = static_cast<int64_t>(exp10(static_cast<double>(-scale)));
    auto const num_zeros = std::max(0, (-scale - count_digits(abs_value % exp_ten)));

    d_buffer += integer_to_string(abs_value / exp_ten, d_buffer);  // add the integer part
    *d_buffer++ = '.';                                             // add decimal point

    thrust::generate_n(thrust::seq, d_buffer, num_zeros, []() { return '0'; });  // add zeros
    d_buffer += num_zeros;

    integer_to_string(abs_value % exp_ten, d_buffer);  // add the fraction part
  }
};

/**
 * @brief The dispatcher functor for converting fixed-point values into strings.
 */
struct dispatch_from_fixed_point_fn {
  template <typename T, std::enable_if_t<cudf::is_fixed_point<T>()>* = nullptr>
  std::unique_ptr<column> operator()(column_view const& input,
                                     rmm::cuda_stream_view stream,
                                     rmm::mr::device_memory_resource* mr) const
  {
    using DecimalType = device_storage_type_t<T>;  // underlying value type

    auto const d_column = column_device_view::create(input, stream);

    // build offsets column
    auto offsets_transformer_itr = cudf::detail::make_counting_transform_iterator(
      0, decimal_to_string_size_fn<DecimalType>{*d_column});
    auto offsets_column = detail::make_offsets_child_column(
      offsets_transformer_itr, offsets_transformer_itr + input.size(), stream, mr);
    auto const d_offsets = offsets_column->view().template data<int32_t>();

    // build chars column
    auto const bytes =
      cudf::detail::get_value<int32_t>(offsets_column->view(), input.size(), stream);
    auto chars_column = detail::create_chars_child_column(input.size(), bytes, stream, mr);
    auto d_chars      = chars_column->mutable_view().template data<char>();
    thrust::for_each_n(rmm::exec_policy(stream),
                       thrust::make_counting_iterator<size_type>(0),
                       input.size(),
                       decimal_to_string_fn<DecimalType>{*d_column, d_offsets, d_chars});

    return make_strings_column(input.size(),
                               std::move(offsets_column),
                               std::move(chars_column),
                               input.null_count(),
                               cudf::detail::copy_bitmask(input, stream, mr),
                               stream,
                               mr);
  }

  template <typename T, std::enable_if_t<not cudf::is_fixed_point<T>()>* = nullptr>
  std::unique_ptr<column> operator()(column_view const&,
                                     rmm::cuda_stream_view,
                                     rmm::mr::device_memory_resource*) const
  {
    CUDF_FAIL("Values for from_fixed_point function must be a decimal type.");
  }
};

}  // namespace

std::unique_ptr<column> from_fixed_point(column_view const& input,
                                         rmm::cuda_stream_view stream,
                                         rmm::mr::device_memory_resource* mr)
{
  if (input.is_empty()) return detail::make_empty_strings_column(stream, mr);
  return type_dispatcher(input.type(), dispatch_from_fixed_point_fn{}, input, stream, mr);
}

}  // namespace detail

// external API

std::unique_ptr<column> from_fixed_point(column_view const& input,
                                         rmm::mr::device_memory_resource* mr)
{
  CUDF_FUNC_RANGE();
  return detail::from_fixed_point(input, rmm::cuda_stream_default, mr);
}

namespace detail {
namespace {

struct dispatch_is_fixed_point_fn {
  template <typename T, std::enable_if_t<cudf::is_fixed_point<T>()>* = nullptr>
  std::unique_ptr<column> operator()(strings_column_view const& input,
                                     data_type decimal_type,
                                     rmm::cuda_stream_view stream,
                                     rmm::mr::device_memory_resource* mr) const
  {
    using DecimalType = device_storage_type_t<T>;

    auto const d_column = column_device_view::create(input.parent(), stream);

    // create output column
    auto results   = make_numeric_column(data_type{type_id::BOOL8},
                                       input.size(),
                                       cudf::detail::copy_bitmask(input.parent(), stream, mr),
                                       input.null_count(),
                                       stream,
                                       mr);
    auto d_results = results->mutable_view().data<bool>();

    // check strings for valid fixed-point chars
    thrust::transform(rmm::exec_policy(stream),
                      thrust::make_counting_iterator<size_type>(0),
                      thrust::make_counting_iterator<size_type>(input.size()),
                      d_results,
                      string_to_decimal_check_fn<DecimalType>{*d_column, decimal_type.scale()});
    results->set_null_count(input.null_count());
    return results;
  }

  template <typename T, std::enable_if_t<not cudf::is_fixed_point<T>()>* = nullptr>
  std::unique_ptr<column> operator()(strings_column_view const&,
                                     data_type,
                                     rmm::cuda_stream_view,
                                     rmm::mr::device_memory_resource*) const
  {
    CUDF_FAIL("is_fixed_point is expecting a decimal type");
  }
};

}  // namespace

std::unique_ptr<column> is_fixed_point(strings_column_view const& input,
                                       data_type decimal_type,
                                       rmm::cuda_stream_view stream,
                                       rmm::mr::device_memory_resource* mr)
{
  if (input.is_empty()) return cudf::make_empty_column(data_type{type_id::BOOL8});
  return type_dispatcher(
    decimal_type, dispatch_is_fixed_point_fn{}, input, decimal_type, stream, mr);
}
}  // namespace detail

std::unique_ptr<column> is_fixed_point(strings_column_view const& input,
                                       data_type decimal_type,
                                       rmm::mr::device_memory_resource* mr)
{
  CUDF_FUNC_RANGE();
  return detail::is_fixed_point(input, decimal_type, rmm::cuda_stream_default, mr);
}

}  // namespace strings
}  // namespace cudf
