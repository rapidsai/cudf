/*
 * Copyright (c) 2019-2020, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <cudf/column/column_device_view.cuh>
#include <cudf/column/column_factories.hpp>
#include <cudf/detail/nvtx/ranges.hpp>
#include <cudf/strings/convert/convert_ipv4.hpp>
#include <cudf/strings/detail/utilities.hpp>
#include <cudf/strings/string_view.cuh>
#include <cudf/strings/strings_column_view.hpp>
#include <strings/utilities.cuh>

#include <thrust/transform.h>

namespace cudf {
namespace strings {
namespace detail {
namespace {
/**
 * @brief Converts IPv4 strings into integers.
 *
 * Only single-byte characters are expected.
 * No checking is done on the format of individual strings.
 * Any character that is not [0-9] is considered a delimiter.
 * This means "128-34-56-709" will parse successfully.
 */
struct ipv4_to_integers_fn {
  column_device_view const d_strings;

  __device__ int64_t operator()(size_type idx)
  {
    if (d_strings.is_null(idx)) return 0;
    string_view d_str  = d_strings.element<string_view>(idx);
    uint32_t ipvals[4] = {0};  // IPV4 format: xxx.xxx.xxx.xxx
    int32_t ipv_idx    = 0;
    int32_t factor     = 1;
    const char* in_ptr = d_str.data();
    const char* end    = in_ptr + d_str.size_bytes();
    while ((in_ptr < end) && (ipv_idx < 4)) {
      char ch = *in_ptr++;
      if (ch < '0' || ch > '9') {
        ++ipv_idx;
        factor = 1;
      } else {
        ipvals[ipv_idx] = (ipvals[ipv_idx] * factor) + static_cast<uint32_t>(ch - '0');
        factor          = 10;
      }
    }
    uint32_t result = (ipvals[0] << 24) + (ipvals[1] << 16) + (ipvals[2] << 8) + ipvals[3];
    return static_cast<int64_t>(result);
  }
};

}  // namespace

// Convert strings column of IPv4 addresses to integers column
std::unique_ptr<column> ipv4_to_integers(
  strings_column_view const& strings,
  rmm::mr::device_memory_resource* mr = rmm::mr::get_default_resource(),
  hipStream_t stream                 = 0)
{
  size_type strings_count = strings.size();
  if (strings_count == 0) return make_numeric_column(data_type{type_id::INT64}, 0);

  auto strings_column = column_device_view::create(strings.parent(), stream);
  // create output column copying the strings' null-mask
  auto results   = make_numeric_column(data_type{type_id::INT64},
                                     strings_count,
                                     copy_bitmask(strings.parent(), stream, mr),
                                     strings.null_count(),
                                     stream,
                                     mr);
  auto d_results = results->mutable_view().data<int64_t>();
  // fill output column with ipv4 integers
  thrust::transform(rmm::exec_policy(stream)->on(stream),
                    thrust::make_counting_iterator<size_type>(0),
                    thrust::make_counting_iterator<size_type>(strings_count),
                    d_results,
                    ipv4_to_integers_fn{*strings_column});
  // done
  results->set_null_count(strings.null_count());
  return results;
}

}  // namespace detail

// external API
std::unique_ptr<column> ipv4_to_integers(strings_column_view const& strings,
                                         rmm::mr::device_memory_resource* mr)
{
  CUDF_FUNC_RANGE();
  return detail::ipv4_to_integers(strings, mr);
}

namespace detail {
namespace {
/**
 * @brief Converts integers into IPv4 addresses.
 *
 * Each integer is divided into 8-bit sub-integers.
 * The sub-integers are converted into 1-3 character digits.
 * These are placed appropriately between '.' character.
 */
struct integers_to_ipv4_fn {
  column_device_view const d_column;
  int32_t const* d_offsets;
  char* d_chars;

  __device__ int convert(int value, char* digits)
  {
    int digits_idx = 0;
    while ((value > 0) && (digits_idx < 3)) {
      digits[digits_idx++] = '0' + (value % 10);
      value                = value / 10;
    }
    return digits_idx;
  }

  __device__ void operator()(size_type idx)
  {
    if (d_column.is_null(idx)) return;
    int64_t ip_number = d_column.element<int64_t>(idx);
    char* out_ptr     = d_chars + d_offsets[idx];
    int shift_bits    = 24;
    for (int n = 0; n < 4; ++n) {
      int value = static_cast<int>((ip_number >> shift_bits) & 0x00FF);
      if (value == 0)
        *out_ptr++ = '0';
      else {
        char digits[3];
        int num_digits = convert(value, digits);
        while (num_digits-- > 0) *out_ptr++ = digits[num_digits];
      }
      if ((n + 1) < 4) *out_ptr++ = '.';
      shift_bits -= 8;
    }
  }
};

}  // namespace

// Convert integers into IPv4 addresses
std::unique_ptr<column> integers_to_ipv4(
  column_view const& integers,
  rmm::mr::device_memory_resource* mr = rmm::mr::get_default_resource(),
  hipStream_t stream                 = 0)
{
  size_type strings_count = integers.size();
  if (strings_count == 0) return make_empty_strings_column(mr, stream);

  CUDF_EXPECTS(integers.type().id() == type_id::INT64, "Input column must be type_id::INT64 type");

  auto column   = column_device_view::create(integers, stream);
  auto d_column = *column;

  // copy null mask
  rmm::device_buffer null_mask = copy_bitmask(integers, stream, mr);
  // build offsets column
  auto offsets_transformer_itr = thrust::make_transform_iterator(
    thrust::make_counting_iterator<int32_t>(0), [d_column] __device__(size_type idx) {
      if (d_column.is_null(idx)) return 0;
      size_type bytes   = 3;  // at least 3 dots: xxx.xxx.xxx.xxx
      int64_t ip_number = d_column.element<int64_t>(idx);
      for (int n = 0; n < 4; ++n) {
        auto value = ip_number & 0x00FF;
        bytes += (value < 10 ? 1 : (value < 100 ? 2 : 3));
        ip_number = ip_number >> 8;
      }
      return bytes;
    });
  auto offsets_column = make_offsets_child_column(
    offsets_transformer_itr, offsets_transformer_itr + strings_count, mr, stream);
  auto d_offsets = offsets_column->view().data<int32_t>();

  // build chars column
  size_type bytes = thrust::device_pointer_cast(d_offsets)[strings_count];
  auto chars_column =
    create_chars_child_column(strings_count, integers.null_count(), bytes, mr, stream);
  auto d_chars = chars_column->mutable_view().data<char>();
  thrust::for_each_n(rmm::exec_policy(stream)->on(stream),
                     thrust::make_counting_iterator<size_type>(0),
                     strings_count,
                     integers_to_ipv4_fn{d_column, d_offsets, d_chars});
  //
  return make_strings_column(strings_count,
                             std::move(offsets_column),
                             std::move(chars_column),
                             integers.null_count(),
                             std::move(null_mask),
                             stream,
                             mr);
}

}  // namespace detail

// external API

std::unique_ptr<column> integers_to_ipv4(column_view const& integers,
                                         rmm::mr::device_memory_resource* mr)
{
  CUDF_FUNC_RANGE();
  return detail::integers_to_ipv4(integers, mr);
}

}  // namespace strings
}  // namespace cudf
