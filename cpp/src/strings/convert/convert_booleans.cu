/*
 * Copyright (c) 2019-2021, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <cudf/column/column_device_view.cuh>
#include <cudf/column/column_factories.hpp>
#include <cudf/detail/iterator.cuh>
#include <cudf/detail/null_mask.hpp>
#include <cudf/detail/nvtx/ranges.hpp>
#include <cudf/strings/convert/convert_booleans.hpp>
#include <cudf/strings/detail/converters.hpp>
#include <cudf/strings/detail/utilities.hpp>
#include <cudf/strings/string_view.cuh>
#include <cudf/strings/strings_column_view.hpp>
#include <cudf/utilities/traits.hpp>
#include <cudf/utilities/type_dispatcher.hpp>

#include <strings/utilities.cuh>

#include <rmm/cuda_stream_view.hpp>
#include <rmm/exec_policy.hpp>

#include <thrust/iterator/counting_iterator.h>
#include <thrust/transform.h>

namespace cudf {
namespace strings {
namespace detail {
// Convert strings column to boolean column
std::unique_ptr<column> to_booleans(strings_column_view const& strings,
                                    string_scalar const& true_string,
                                    rmm::cuda_stream_view stream,
                                    rmm::mr::device_memory_resource* mr)
{
  size_type strings_count = strings.size();
  if (strings_count == 0) return make_numeric_column(data_type{type_id::BOOL8}, 0);

  CUDF_EXPECTS(true_string.is_valid() && true_string.size() > 0,
               "Parameter true_string must not be empty.");
  auto d_true = string_view(true_string.data(), true_string.size());

  auto strings_column = column_device_view::create(strings.parent(), stream);
  auto d_strings      = *strings_column;
  // create output column copying the strings' null-mask
  auto results      = make_numeric_column(data_type{type_id::BOOL8},
                                     strings_count,
                                     cudf::detail::copy_bitmask(strings.parent(), stream, mr),
                                     strings.null_count(),
                                     stream,
                                     mr);
  auto results_view = results->mutable_view();
  auto d_results    = results_view.data<bool>();

  thrust::transform(rmm::exec_policy(stream),
                    thrust::make_counting_iterator<size_type>(0),
                    thrust::make_counting_iterator<size_type>(strings_count),
                    d_results,
                    [d_strings, d_true] __device__(size_type idx) {
                      bool result = false;
                      if (!d_strings.is_null(idx))
                        result = d_strings.element<string_view>(idx).compare(d_true) == 0;
                      return result;
                    });
  results->set_null_count(strings.null_count());
  return results;
}

}  // namespace detail

// external API
std::unique_ptr<column> to_booleans(strings_column_view const& strings,
                                    string_scalar const& true_string,
                                    rmm::mr::device_memory_resource* mr)
{
  CUDF_FUNC_RANGE();
  return detail::to_booleans(strings, true_string, rmm::cuda_stream_default, mr);
}

namespace detail {
// Convert boolean column to strings column
std::unique_ptr<column> from_booleans(column_view const& booleans,
                                      string_scalar const& true_string,
                                      string_scalar const& false_string,
                                      rmm::cuda_stream_view stream,
                                      rmm::mr::device_memory_resource* mr)
{
  size_type strings_count = booleans.size();
  if (strings_count == 0) return make_empty_strings_column(stream, mr);

  CUDF_EXPECTS(booleans.type().id() == type_id::BOOL8, "Input column must be boolean type");
  CUDF_EXPECTS(true_string.is_valid() && true_string.size() > 0,
               "Parameter true_string must not be empty.");
  auto d_true = string_view(true_string.data(), true_string.size());
  CUDF_EXPECTS(false_string.is_valid() && false_string.size() > 0,
               "Parameter false_string must not be empty.");
  auto d_false = string_view(false_string.data(), false_string.size());

  auto column   = column_device_view::create(booleans, stream);
  auto d_column = *column;

  // copy null mask
  rmm::device_buffer null_mask = cudf::detail::copy_bitmask(booleans, stream, mr);
  // build offsets column
  auto offsets_transformer_itr = cudf::detail::make_counting_transform_iterator(
    0, [d_column, d_true, d_false] __device__(size_type idx) {
      if (d_column.is_null(idx)) return 0;
      return d_column.element<bool>(idx) ? d_true.size_bytes() : d_false.size_bytes();
    });
  auto offsets_column = make_offsets_child_column(
    offsets_transformer_itr, offsets_transformer_itr + strings_count, stream, mr);
  auto offsets_view = offsets_column->view();
  auto d_offsets    = offsets_view.data<int32_t>();

  // build chars column
  size_type bytes = thrust::device_pointer_cast(d_offsets)[strings_count];
  auto chars_column =
    create_chars_child_column(strings_count, booleans.null_count(), bytes, stream, mr);
  auto chars_view = chars_column->mutable_view();
  auto d_chars    = chars_view.data<char>();
  thrust::for_each_n(rmm::exec_policy(stream),
                     thrust::make_counting_iterator<size_type>(0),
                     strings_count,
                     [d_column, d_true, d_false, d_offsets, d_chars] __device__(size_type idx) {
                       if (d_column.is_null(idx)) return;
                       string_view result = (d_column.element<bool>(idx) ? d_true : d_false);
                       memcpy(d_chars + d_offsets[idx], result.data(), result.size_bytes());
                     });

  return make_strings_column(strings_count,
                             std::move(offsets_column),
                             std::move(chars_column),
                             booleans.null_count(),
                             std::move(null_mask),
                             stream,
                             mr);
}

}  // namespace detail

// external API

std::unique_ptr<column> from_booleans(column_view const& booleans,
                                      string_scalar const& true_string,
                                      string_scalar const& false_string,
                                      rmm::mr::device_memory_resource* mr)
{
  CUDF_FUNC_RANGE();
  return detail::from_booleans(booleans, true_string, false_string, rmm::cuda_stream_default, mr);
}

}  // namespace strings
}  // namespace cudf
