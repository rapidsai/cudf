/*
 * Copyright (c) 2019-2020, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <cudf/column/column_device_view.cuh>
#include <cudf/column/column_factories.hpp>
#include <cudf/detail/null_mask.hpp>
#include <cudf/detail/nvtx/ranges.hpp>
#include <cudf/detail/valid_if.cuh>
#include <cudf/scalar/scalar_device_view.cuh>
#include <cudf/strings/combine.hpp>
#include <cudf/strings/detail/combine.hpp>
#include <cudf/strings/detail/utilities.hpp>
#include <cudf/strings/string_view.cuh>
#include <cudf/strings/strings_column_view.hpp>
#include <cudf/table/table_device_view.cuh>
#include <cudf/utilities/error.hpp>

#include <strings/utilities.cuh>

#include <rmm/thrust_rmm_allocator.h>

#include <thrust/logical.h>
#include <thrust/transform_reduce.h>
#include <thrust/transform_scan.h>

#include <algorithm>

namespace cudf {
namespace strings {
namespace detail {
//
std::unique_ptr<column> concatenate(table_view const& strings_columns,
                                    string_scalar const& separator,
                                    string_scalar const& narep,
                                    rmm::mr::device_memory_resource* mr,
                                    hipStream_t stream = 0)
{
  auto num_columns = strings_columns.num_columns();
  CUDF_EXPECTS(num_columns > 0, "At least one column must be specified");
  // check all columns are of type string
  CUDF_EXPECTS(std::all_of(strings_columns.begin(),
                           strings_columns.end(),
                           [](auto c) { return c.type().id() == type_id::STRING; }),
               "All columns must be of type string");
  if (num_columns == 1)  // single strings column returns a copy
    return std::make_unique<column>(*(strings_columns.begin()), stream, mr);
  auto strings_count = strings_columns.num_rows();
  if (strings_count == 0)  // empty begets empty
    return detail::make_empty_strings_column(mr, stream);

  CUDF_EXPECTS(separator.is_valid(), "Parameter separator must be a valid string_scalar");
  string_view d_separator(separator.data(), separator.size());
  auto d_narep = get_scalar_device_view(const_cast<string_scalar&>(narep));

  // Create device views from the strings columns.
  auto table   = table_device_view::create(strings_columns, stream);
  auto d_table = *table;

  // create resulting null mask
  auto valid_mask = cudf::detail::valid_if(
    thrust::make_counting_iterator<size_type>(0),
    thrust::make_counting_iterator<size_type>(strings_count),
    [d_table, d_narep] __device__(size_type idx) {
      bool null_element = thrust::any_of(
        thrust::seq, d_table.begin(), d_table.end(), [idx](auto col) { return col.is_null(idx); });
      return (!null_element || d_narep.is_valid());
    },
    stream,
    mr);
  auto& null_mask       = valid_mask.first;
  auto const null_count = valid_mask.second;

  // build offsets column by computing sizes of each string in the output
  auto offsets_transformer = [d_table, num_columns, d_separator, d_narep] __device__(
                               size_type row_idx) {
    // for this row (idx), iterate over each column and add up the bytes
    bool null_element =
      thrust::any_of(thrust::seq, d_table.begin(), d_table.end(), [row_idx](auto const& d_column) {
        return d_column.is_null(row_idx);
      });
    if (null_element && !d_narep.is_valid()) return 0;
    size_type bytes = thrust::transform_reduce(
      thrust::seq,
      d_table.begin(),
      d_table.end(),
      [row_idx, d_separator, d_narep] __device__(column_device_view const& d_column) {
        return d_separator.size_bytes() + (d_column.is_null(row_idx)
                                             ? d_narep.size()
                                             : d_column.element<string_view>(row_idx).size_bytes());
      },
      0,
      thrust::plus<size_type>());
    // separator goes only in between elements
    if (bytes > 0)                        // if not null
      bytes -= d_separator.size_bytes();  // remove the last separator
    return bytes;
  };
  auto offsets_transformer_itr = thrust::make_transform_iterator(
    thrust::make_counting_iterator<size_type>(0), offsets_transformer);
  auto offsets_column = detail::make_offsets_child_column(
    offsets_transformer_itr, offsets_transformer_itr + strings_count, mr, stream);
  auto d_results_offsets = offsets_column->view().data<int32_t>();

  // create the chars column
  size_type bytes = thrust::device_pointer_cast(d_results_offsets)[strings_count];
  auto chars_column =
    strings::detail::create_chars_child_column(strings_count, null_count, bytes, mr, stream);
  // fill the chars column
  auto d_results_chars = chars_column->mutable_view().data<char>();
  thrust::for_each_n(
    rmm::exec_policy(stream)->on(stream),
    thrust::make_counting_iterator<size_type>(0),
    strings_count,
    [d_table, num_columns, d_separator, d_narep, d_results_offsets, d_results_chars] __device__(
      size_type idx) {
      bool null_element = thrust::any_of(
        thrust::seq, d_table.begin(), d_table.end(), [idx](column_device_view const& col) {
          return col.is_null(idx);
        });
      if (null_element && !d_narep.is_valid())
        return;  // do not write to buffer at all if any column element for this row is null
      size_type offset = d_results_offsets[idx];
      char* d_buffer   = d_results_chars + offset;
      // write out each column's entry for this row
      for (size_type col_idx = 0; col_idx < num_columns; ++col_idx) {
        auto d_column = d_table.column(col_idx);
        string_view d_str =
          d_column.is_null(idx) ? d_narep.value() : d_column.element<string_view>(idx);
        d_buffer = detail::copy_string(d_buffer, d_str);
        // separator goes only in between elements
        if (col_idx + 1 < num_columns) d_buffer = detail::copy_string(d_buffer, d_separator);
      }
    });

  return make_strings_column(strings_count,
                             std::move(offsets_column),
                             std::move(chars_column),
                             null_count,
                             std::move(null_mask),
                             stream,
                             mr);
}

//
std::unique_ptr<column> join_strings(strings_column_view const& strings,
                                     string_scalar const& separator,
                                     string_scalar const& narep,
                                     rmm::mr::device_memory_resource* mr,
                                     hipStream_t stream = 0)
{
  auto strings_count = strings.size();
  if (strings_count == 0) return detail::make_empty_strings_column(mr, stream);

  CUDF_EXPECTS(separator.is_valid(), "Parameter separator must be a valid string_scalar");

  auto execpol = rmm::exec_policy(stream);
  string_view d_separator(separator.data(), separator.size());
  auto d_narep = get_scalar_device_view(const_cast<string_scalar&>(narep));

  auto strings_column = column_device_view::create(strings.parent(), stream);
  auto d_strings      = *strings_column;

  // create an offsets array for building the output memory layout
  rmm::device_vector<size_type> output_offsets(strings_count + 1);
  auto d_output_offsets = output_offsets.data().get();
  // using inclusive-scan to compute last entry which is the total size
  thrust::transform_inclusive_scan(
    execpol->on(stream),
    thrust::make_counting_iterator<size_type>(0),
    thrust::make_counting_iterator<size_type>(strings_count),
    d_output_offsets + 1,
    [d_strings, d_separator, d_narep] __device__(size_type idx) {
      size_type bytes = 0;
      if (d_strings.is_null(idx)) {
        if (!d_narep.is_valid()) return 0;  // skip nulls
        bytes += d_narep.size();
      } else
        bytes += d_strings.element<string_view>(idx).size_bytes();
      if ((idx + 1) < d_strings.size()) bytes += d_separator.size_bytes();
      return bytes;
    },
    thrust::plus<size_type>());
  CUDA_TRY(hipMemsetAsync(d_output_offsets, 0, sizeof(size_type), stream));
  // total size is the last entry
  size_type bytes = output_offsets.back();

  // build offsets column (only 1 string so 2 offset entries)
  auto offsets_column =
    make_numeric_column(data_type{type_id::INT32}, 2, mask_state::UNALLOCATED, stream, mr);
  auto offsets_view = offsets_column->mutable_view();
  // set the first entry to 0 and the last entry to bytes
  int32_t new_offsets[] = {0, bytes};
  CUDA_TRY(hipMemcpyAsync(offsets_view.data<int32_t>(),
                           new_offsets,
                           sizeof(new_offsets),
                           hipMemcpyHostToDevice,
                           stream));

  // build null mask
  // only one entry so it is either all valid or all null
  size_type null_count = 0;
  rmm::device_buffer null_mask{0, stream, mr};  // init to null null-mask
  if (strings.null_count() == strings_count && !narep.is_valid()) {
    null_mask  = cudf::detail::create_null_mask(1, cudf::mask_state::ALL_NULL, stream, mr);
    null_count = 1;
  }
  auto chars_column =
    detail::create_chars_child_column(strings_count, null_count, bytes, mr, stream);
  auto chars_view = chars_column->mutable_view();
  auto d_chars    = chars_view.data<char>();
  thrust::for_each_n(
    execpol->on(stream),
    thrust::make_counting_iterator<size_type>(0),
    strings_count,
    [d_strings, d_separator, d_narep, d_output_offsets, d_chars] __device__(size_type idx) {
      size_type offset = d_output_offsets[idx];
      char* d_buffer   = d_chars + offset;
      if (d_strings.is_null(idx)) {
        if (!d_narep.is_valid())
          return;  // do not write to buffer if element is null (including separator)
        d_buffer = detail::copy_string(d_buffer, d_narep.value());
      } else {
        string_view d_str = d_strings.element<string_view>(idx);
        d_buffer          = detail::copy_string(d_buffer, d_str);
      }
      if ((idx + 1) < d_strings.size()) d_buffer = detail::copy_string(d_buffer, d_separator);
    });

  return make_strings_column(1,
                             std::move(offsets_column),
                             std::move(chars_column),
                             null_count,
                             std::move(null_mask),
                             stream,
                             mr);
}

//
std::unique_ptr<column> concatenate(table_view const& strings_columns,
                                    strings_column_view const& separators,
                                    string_scalar const& separator_narep,
                                    string_scalar const& col_narep,
                                    rmm::mr::device_memory_resource* mr,
                                    hipStream_t stream = 0)
{
  auto num_columns = strings_columns.num_columns();
  CUDF_EXPECTS(num_columns > 0, "At least one column must be specified");
  // Check if all columns are of type string
  CUDF_EXPECTS(std::all_of(strings_columns.begin(),
                           strings_columns.end(),
                           [](auto c) { return c.type().id() == type_id::STRING; }),
               "All columns must be of type string");

  auto strings_count = strings_columns.num_rows();
  CUDF_EXPECTS(strings_count == separators.size(),
               "Separators column should be the same size as the strings columns");
  if (strings_count == 0)  // Empty begets empty
    return detail::make_empty_strings_column(mr, stream);

  // Invalid output column strings - null rows
  string_view const invalid_str{nullptr, 0};
  auto const separator_rep = get_scalar_device_view(const_cast<string_scalar&>(separator_narep));
  auto const col_rep       = get_scalar_device_view(const_cast<string_scalar&>(col_narep));
  auto const separator_col_view_ptr = column_device_view::create(separators.parent(), stream);
  auto const separator_col_view     = *separator_col_view_ptr;

  if (num_columns == 1) {
    // Shallow copy of the resultant strings
    rmm::device_vector<string_view> out_col_strings(strings_count);

    // Device view of the only column in the table view
    auto const col0_ptr = column_device_view::create(strings_columns.column(0), stream);
    auto const col0     = *col0_ptr;

    // Execute it on every element
    thrust::transform(
      rmm::exec_policy(stream)->on(stream),
      thrust::make_counting_iterator(0),
      thrust::make_counting_iterator(strings_count),
      out_col_strings.data().get(),
      // Output depends on the separator
      [col0, invalid_str, separator_col_view, separator_rep, col_rep] __device__(auto ridx) {
        if (!separator_col_view.is_valid(ridx) && !separator_rep.is_valid()) return invalid_str;
        if (col0.is_valid(ridx)) {
          auto sv = col0.element<string_view>(ridx);
          return sv.empty() ? string_view{} : sv;
        } else if (col_rep.is_valid()) {
          auto cv = col_rep.value();
          return cv.empty() ? string_view{} : cv;
        } else
          return invalid_str;
      });

    return make_strings_column(out_col_strings, invalid_str, stream, mr);
  }

  // Create device views from the strings columns.
  auto table   = table_device_view::create(strings_columns, stream);
  auto d_table = *table;

  // Create resulting null mask
  auto valid_mask = cudf::detail::valid_if(
    thrust::make_counting_iterator<size_type>(0),
    thrust::make_counting_iterator<size_type>(strings_count),
    [d_table, separator_col_view, separator_rep, col_rep] __device__(size_type ridx) {
      if (!separator_col_view.is_valid(ridx) && !separator_rep.is_valid()) return false;
      bool all_nulls =
        thrust::all_of(thrust::seq, d_table.begin(), d_table.end(), [ridx](auto const& col) {
          return col.is_null(ridx);
        });
      return all_nulls ? col_rep.is_valid() : true;
    },
    stream,
    mr);

  auto null_count = valid_mask.second;

  // Build offsets column by computing sizes of each string in the output
  auto offsets_transformer = [d_table, separator_col_view, separator_rep, col_rep] __device__(
                               size_type ridx) {
    // If the separator value for the row is null and if there aren't global separator
    // replacements, this row does not have any value - null row
    if (!separator_col_view.is_valid(ridx) && !separator_rep.is_valid()) return 0;

    // For this row (idx), iterate over each column and add up the bytes
    bool all_nulls =
      thrust::all_of(thrust::seq, d_table.begin(), d_table.end(), [ridx](auto const& d_column) {
        return d_column.is_null(ridx);
      });
    // If all column values are null and there isn't a global column replacement value, this row
    // is a null row
    if (all_nulls && !col_rep.is_valid()) return 0;

    // There is at least one non-null column value (it can still be empty though)
    auto separator_str = separator_col_view.is_valid(ridx)
                           ? separator_col_view.element<string_view>(ridx)
                           : separator_rep.value();

    size_type bytes = thrust::transform_reduce(
      thrust::seq,
      d_table.begin(),
      d_table.end(),
      [ridx, separator_str, col_rep] __device__(column_device_view const& d_column) {
        // If column is null and there isn't a valid column replacement, this isn't used in
        // final string concatenate
        if (d_column.is_null(ridx) && !col_rep.is_valid()) return 0;
        return separator_str.size_bytes() + (d_column.is_null(ridx)
                                               ? col_rep.size()
                                               : d_column.element<string_view>(ridx).size_bytes());
      },
      0,
      thrust::plus<size_type>());

    // Null/empty separator and columns doesn't produce a non-empty string
    if (bytes == 0) assert(separator_str.size_bytes() == 0);

    // Separator goes only in between elements
    return bytes - separator_str.size_bytes();
  };
  auto offsets_transformer_itr = thrust::make_transform_iterator(
    thrust::make_counting_iterator<size_type>(0), offsets_transformer);
  auto offsets_column = detail::make_offsets_child_column(
    offsets_transformer_itr, offsets_transformer_itr + strings_count, mr, stream);
  auto d_results_offsets = offsets_column->view().data<int32_t>();

  // Create the chars column
  size_type bytes = thrust::device_pointer_cast(d_results_offsets)[strings_count];
  auto chars_column =
    strings::detail::create_chars_child_column(strings_count, null_count, bytes, mr, stream);

  // Fill the chars column
  auto d_results_chars = chars_column->mutable_view().data<char>();
  thrust::for_each_n(rmm::exec_policy(stream)->on(stream),
                     thrust::make_counting_iterator<size_type>(0),
                     strings_count,
                     [d_table,
                      num_columns,
                      d_results_offsets,
                      d_results_chars,
                      separator_col_view,
                      separator_rep,
                      col_rep] __device__(size_type ridx) {
                       // If the separator for this row is null and if there isn't a valid separator
                       // to replace, do not write anything for this row
                       if (!separator_col_view.is_valid(ridx) && !separator_rep.is_valid()) return;

                       bool all_nulls = thrust::all_of(
                         thrust::seq, d_table.begin(), d_table.end(), [ridx](auto const& col) {
                           return col.is_null(ridx);
                         });

                       // If all column values are null and there isn't a valid column replacement,
                       // skip this row
                       if (all_nulls && !col_rep.is_valid()) return;

                       size_type offset    = d_results_offsets[ridx];
                       char* d_buffer      = d_results_chars + offset;
                       bool colval_written = false;

                       // There is at least one non-null column value (it can still be empty though)
                       auto separator_str = separator_col_view.is_valid(ridx)
                                              ? separator_col_view.element<string_view>(ridx)
                                              : separator_rep.value();

                       // Write out each column's entry for this row
                       for (size_type col_idx = 0; col_idx < num_columns; ++col_idx) {
                         auto d_column = d_table.column(col_idx);
                         // If the column isn't valid and if there isn't a replacement for it, skip
                         // it
                         if (d_column.is_null(ridx) && !col_rep.is_valid()) continue;

                         // Separator goes only in between elements
                         if (colval_written)
                           d_buffer = detail::copy_string(d_buffer, separator_str);

                         string_view d_str = d_column.is_null(ridx)
                                               ? col_rep.value()
                                               : d_column.element<string_view>(ridx);
                         d_buffer       = detail::copy_string(d_buffer, d_str);
                         colval_written = true;
                       }
                     });

  return make_strings_column(strings_count,
                             std::move(offsets_column),
                             std::move(chars_column),
                             null_count,
                             (null_count) ? std::move(valid_mask.first) : rmm::device_buffer{},
                             stream,
                             mr);
}

}  // namespace detail

// APIs

std::unique_ptr<column> concatenate(table_view const& strings_columns,
                                    string_scalar const& separator,
                                    string_scalar const& narep,
                                    rmm::mr::device_memory_resource* mr)
{
  CUDF_FUNC_RANGE();
  return detail::concatenate(strings_columns, separator, narep, mr);
}

std::unique_ptr<column> join_strings(strings_column_view const& strings,
                                     string_scalar const& separator,
                                     string_scalar const& narep,
                                     rmm::mr::device_memory_resource* mr)
{
  CUDF_FUNC_RANGE();
  return detail::join_strings(strings, separator, narep, mr);
}

std::unique_ptr<column> concatenate(table_view const& strings_columns,
                                    strings_column_view const& separators,
                                    string_scalar const& separator_narep,
                                    string_scalar const& col_narep,
                                    rmm::mr::device_memory_resource* mr)
{
  CUDF_FUNC_RANGE();
  return detail::concatenate(strings_columns, separators, separator_narep, col_narep, mr);
}

}  // namespace strings
}  // namespace cudf
