/*
 * Copyright (c) 2019-2021, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <strings/utilities.cuh>

#include <cudf/column/column_device_view.cuh>
#include <cudf/column/column_factories.hpp>
#include <cudf/detail/get_value.cuh>
#include <cudf/detail/null_mask.hpp>
#include <cudf/detail/nvtx/ranges.hpp>
#include <cudf/detail/valid_if.cuh>
#include <cudf/scalar/scalar_device_view.cuh>
#include <cudf/strings/combine.hpp>
#include <cudf/strings/detail/combine.hpp>
#include <cudf/strings/detail/utilities.hpp>
#include <cudf/strings/strings_column_view.hpp>
#include <cudf/table/table_device_view.cuh>
#include <cudf/utilities/error.hpp>

#include <rmm/cuda_stream_view.hpp>
#include <rmm/device_uvector.hpp>
#include <rmm/exec_policy.hpp>

#include <thrust/iterator/transform_iterator.h>
#include <thrust/logical.h>
#include <thrust/transform_reduce.h>
#include <thrust/transform_scan.h>

#include <cudf/unary.hpp>
#include <cudf_test/column_utilities.hpp>

namespace cudf {
namespace strings {
namespace detail {

std::unique_ptr<column> concatenate(table_view const& strings_columns,
                                    string_scalar const& separator,
                                    string_scalar const& narep,
                                    rmm::cuda_stream_view stream,
                                    rmm::mr::device_memory_resource* mr)
{
  auto const num_columns = strings_columns.num_columns();
  CUDF_EXPECTS(num_columns > 0, "At least one column must be specified");
  // check all columns are of type string
  CUDF_EXPECTS(std::all_of(strings_columns.begin(),
                           strings_columns.end(),
                           [](auto c) { return c.type().id() == type_id::STRING; }),
               "All columns must be of type string");
  if (num_columns == 1)  // single strings column returns a copy
    return std::make_unique<column>(*(strings_columns.begin()), stream, mr);
  auto const strings_count = strings_columns.num_rows();
  if (strings_count == 0)  // empty begets empty
    return detail::make_empty_strings_column(stream, mr);

  CUDF_EXPECTS(separator.is_valid(), "Parameter separator must be a valid string_scalar");
  string_view d_separator(separator.data(), separator.size());
  auto d_narep = get_scalar_device_view(const_cast<string_scalar&>(narep));

  // Create device views from the strings columns.
  auto table   = table_device_view::create(strings_columns, stream);
  auto d_table = *table;

  // create resulting null mask
  auto valid_mask = cudf::detail::valid_if(
    thrust::make_counting_iterator<size_type>(0),
    thrust::make_counting_iterator<size_type>(strings_count),
    [d_table, d_narep] __device__(size_type idx) {
      bool null_element = thrust::any_of(
        thrust::seq, d_table.begin(), d_table.end(), [idx](auto col) { return col.is_null(idx); });
      return (!null_element || d_narep.is_valid());
    },
    stream,
    mr);
  auto& null_mask       = valid_mask.first;
  auto const null_count = valid_mask.second;

  // build offsets column by computing sizes of each string in the output
  auto offsets_transformer = [d_table, d_separator, d_narep] __device__(size_type row_idx) {
    // for this row (idx), iterate over each column and add up the bytes
    bool const null_element =
      thrust::any_of(thrust::seq, d_table.begin(), d_table.end(), [row_idx](auto const& d_column) {
        return d_column.is_null(row_idx);
      });
    if (null_element && !d_narep.is_valid()) return 0;
    size_type const bytes = thrust::transform_reduce(
      thrust::seq,
      d_table.begin(),
      d_table.end(),
      [row_idx, d_separator, d_narep] __device__(column_device_view const& d_column) {
        return d_separator.size_bytes() + (d_column.is_null(row_idx)
                                             ? d_narep.size()
                                             : d_column.element<string_view>(row_idx).size_bytes());
      },
      0,
      thrust::plus<size_type>());
    // separator goes only in between elements
    return bytes == 0 ? 0 : (bytes - d_separator.size_bytes());  // remove the last separator
  };
  auto offsets_transformer_itr = thrust::make_transform_iterator(
    thrust::make_counting_iterator<size_type>(0), offsets_transformer);
  auto offsets_column = detail::make_offsets_child_column(
    offsets_transformer_itr, offsets_transformer_itr + strings_count, stream, mr);
  auto d_results_offsets = offsets_column->view().data<int32_t>();

  // create the chars column
  auto const bytes =
    cudf::detail::get_value<int32_t>(offsets_column->view(), strings_count, stream);
  auto chars_column =
    strings::detail::create_chars_child_column(strings_count, null_count, bytes, stream, mr);
  // fill the chars column
  auto d_results_chars = chars_column->mutable_view().data<char>();
  thrust::for_each_n(
    rmm::exec_policy(stream),
    thrust::make_counting_iterator<size_type>(0),
    strings_count,
    [d_table, num_columns, d_separator, d_narep, d_results_offsets, d_results_chars] __device__(
      size_type idx) {
      bool const null_element = thrust::any_of(
        thrust::seq, d_table.begin(), d_table.end(), [idx](column_device_view const& col) {
          return col.is_null(idx);
        });
      if (null_element && !d_narep.is_valid())
        return;  // do not write to buffer at all if any column element for this row is null
      char* d_buffer = d_results_chars + d_results_offsets[idx];
      // write out each column's entry for this row
      for (size_type col_idx = 0; col_idx < num_columns; ++col_idx) {
        auto const d_column = d_table.column(col_idx);
        string_view const d_str =
          d_column.is_null(idx) ? d_narep.value() : d_column.element<string_view>(idx);
        d_buffer = detail::copy_string(d_buffer, d_str);
        // separator goes only in between elements
        if (col_idx + 1 < num_columns) d_buffer = detail::copy_string(d_buffer, d_separator);
      }
    });

  return make_strings_column(strings_count,
                             std::move(offsets_column),
                             std::move(chars_column),
                             null_count,
                             std::move(null_mask),
                             stream,
                             mr);
}

std::unique_ptr<column> join_strings(strings_column_view const& strings,
                                     string_scalar const& separator,
                                     string_scalar const& narep,
                                     rmm::cuda_stream_view stream,
                                     rmm::mr::device_memory_resource* mr)
{
  auto strings_count = strings.size();
  if (strings_count == 0) return detail::make_empty_strings_column(stream, mr);

  CUDF_EXPECTS(separator.is_valid(), "Parameter separator must be a valid string_scalar");

  string_view d_separator(separator.data(), separator.size());
  auto d_narep = get_scalar_device_view(const_cast<string_scalar&>(narep));

  auto strings_column = column_device_view::create(strings.parent(), stream);
  auto d_strings      = *strings_column;

  // create an offsets array for building the output memory layout
  rmm::device_uvector<size_type> output_offsets(strings_count + 1, stream);
  auto d_output_offsets = output_offsets.data();
  // using inclusive-scan to compute last entry which is the total size
  thrust::transform_inclusive_scan(
    rmm::exec_policy(stream),
    thrust::make_counting_iterator<size_type>(0),
    thrust::make_counting_iterator<size_type>(strings_count),
    d_output_offsets + 1,
    [d_strings, d_separator, d_narep] __device__(size_type idx) {
      size_type bytes = 0;
      if (d_strings.is_null(idx)) {
        if (!d_narep.is_valid()) return 0;  // skip nulls
        bytes += d_narep.size();
      } else
        bytes += d_strings.element<string_view>(idx).size_bytes();
      if ((idx + 1) < d_strings.size()) bytes += d_separator.size_bytes();
      return bytes;
    },
    thrust::plus<size_type>());
  size_type const zero = 0;
  output_offsets.set_element_async(0, zero, stream);
  // total size is the last entry
  // Note this call does a synchronize on the stream and thereby also protects the
  // set_element_async parameter from going out of scope before it is used.
  size_type const bytes = output_offsets.back_element(stream);

  // build offsets column (only 1 string so 2 offset entries)
  auto offsets_column =
    make_numeric_column(data_type{type_id::INT32}, 2, mask_state::UNALLOCATED, stream, mr);
  auto offsets_view = offsets_column->mutable_view();
  // set the first entry to 0 and the last entry to bytes
  int32_t new_offsets[] = {0, static_cast<int32_t>(bytes)};
  CUDA_TRY(hipMemcpyAsync(offsets_view.data<int32_t>(),
                           new_offsets,
                           sizeof(new_offsets),
                           hipMemcpyHostToDevice,
                           stream.value()));

  // build null mask
  // only one entry so it is either all valid or all null
  size_type null_count = 0;
  rmm::device_buffer null_mask{0, stream, mr};  // init to null null-mask
  if (strings.null_count() == strings_count && !narep.is_valid()) {
    null_mask  = cudf::detail::create_null_mask(1, cudf::mask_state::ALL_NULL, stream, mr);
    null_count = 1;
  }
  auto chars_column =
    detail::create_chars_child_column(strings_count, null_count, bytes, stream, mr);
  auto chars_view = chars_column->mutable_view();
  auto d_chars    = chars_view.data<char>();
  thrust::for_each_n(
    rmm::exec_policy(stream),
    thrust::make_counting_iterator<size_type>(0),
    strings_count,
    [d_strings, d_separator, d_narep, d_output_offsets, d_chars] __device__(size_type idx) {
      size_type offset = d_output_offsets[idx];
      char* d_buffer   = d_chars + offset;
      if (d_strings.is_null(idx)) {
        if (!d_narep.is_valid())
          return;  // do not write to buffer if element is null (including separator)
        d_buffer = detail::copy_string(d_buffer, d_narep.value());
      } else {
        string_view d_str = d_strings.element<string_view>(idx);
        d_buffer          = detail::copy_string(d_buffer, d_str);
      }
      if ((idx + 1) < d_strings.size()) d_buffer = detail::copy_string(d_buffer, d_separator);
    });

  return make_strings_column(1,
                             std::move(offsets_column),
                             std::move(chars_column),
                             null_count,
                             std::move(null_mask),
                             stream,
                             mr);
}

std::unique_ptr<column> concatenate(table_view const& strings_columns,
                                    strings_column_view const& separators,
                                    string_scalar const& separator_narep,
                                    string_scalar const& col_narep,
                                    rmm::cuda_stream_view stream,
                                    rmm::mr::device_memory_resource* mr)
{
  auto const num_columns = strings_columns.num_columns();
  CUDF_EXPECTS(num_columns > 0, "At least one column must be specified");
  // Check if all columns are of type string
  CUDF_EXPECTS(std::all_of(strings_columns.begin(),
                           strings_columns.end(),
                           [](auto c) { return c.type().id() == type_id::STRING; }),
               "All columns must be of type string");

  auto const strings_count = strings_columns.num_rows();
  CUDF_EXPECTS(strings_count == separators.size(),
               "Separators column should be the same size as the strings columns");
  if (strings_count == 0)  // Empty begets empty
    return detail::make_empty_strings_column(stream, mr);

  // Invalid output column strings - null rows
  string_view const invalid_str{nullptr, 0};
  auto const separator_rep = get_scalar_device_view(const_cast<string_scalar&>(separator_narep));
  auto const col_rep       = get_scalar_device_view(const_cast<string_scalar&>(col_narep));
  auto const separator_col_view_ptr = column_device_view::create(separators.parent(), stream);
  auto const separator_col_view     = *separator_col_view_ptr;

  if (num_columns == 1) {
    // Shallow copy of the resultant strings
    rmm::device_uvector<string_view> out_col_strings(strings_count, stream);

    // Device view of the only column in the table view
    auto const col0_ptr = column_device_view::create(strings_columns.column(0), stream);
    auto const col0     = *col0_ptr;

    // Execute it on every element
    thrust::transform(
      rmm::exec_policy(stream),
      thrust::make_counting_iterator<size_type>(0),
      thrust::make_counting_iterator<size_type>(strings_count),
      out_col_strings.begin(),
      // Output depends on the separator
      [col0, invalid_str, separator_col_view, separator_rep, col_rep] __device__(auto ridx) {
        if (!separator_col_view.is_valid(ridx) && !separator_rep.is_valid()) return invalid_str;
        if (col0.is_valid(ridx)) {
          auto sv = col0.element<string_view>(ridx);
          return sv.empty() ? string_view{} : sv;
        } else if (col_rep.is_valid()) {
          auto cv = col_rep.value();
          return cv.empty() ? string_view{} : cv;
        } else
          return invalid_str;
      });

    return make_strings_column(out_col_strings, invalid_str, stream, mr);
  }

  // Create device views from the strings columns.
  auto table   = table_device_view::create(strings_columns, stream);
  auto d_table = *table;

  // Create resulting null mask
  auto valid_mask = cudf::detail::valid_if(
    thrust::make_counting_iterator<size_type>(0),
    thrust::make_counting_iterator<size_type>(strings_count),
    [d_table, separator_col_view, separator_rep, col_rep] __device__(size_type ridx) {
      if (!separator_col_view.is_valid(ridx) && !separator_rep.is_valid()) return false;
      bool all_nulls =
        thrust::all_of(thrust::seq, d_table.begin(), d_table.end(), [ridx](auto const& col) {
          return col.is_null(ridx);
        });
      return all_nulls ? col_rep.is_valid() : true;
    },
    stream,
    mr);

  auto null_count = valid_mask.second;

  // Build offsets column by computing sizes of each string in the output
  auto offsets_transformer = [d_table, separator_col_view, separator_rep, col_rep] __device__(
                               size_type ridx) {
    // If the separator value for the row is null and if there aren't global separator
    // replacements, this row does not have any value - null row
    if (!separator_col_view.is_valid(ridx) && !separator_rep.is_valid()) return 0;

    // For this row (idx), iterate over each column and add up the bytes
    bool const all_nulls =
      thrust::all_of(thrust::seq, d_table.begin(), d_table.end(), [ridx](auto const& d_column) {
        return d_column.is_null(ridx);
      });
    // If all column values are null and there isn't a global column replacement value, this row
    // is a null row
    if (all_nulls && !col_rep.is_valid()) return 0;

    // There is at least one non-null column value (it can still be empty though)
    auto const separator_str = separator_col_view.is_valid(ridx)
                                 ? separator_col_view.element<string_view>(ridx)
                                 : separator_rep.value();

    size_type const bytes = thrust::transform_reduce(
      thrust::seq,
      d_table.begin(),
      d_table.end(),
      [ridx, separator_str, col_rep] __device__(column_device_view const& d_column) {
        // If column is null and there isn't a valid column replacement, this isn't used in
        // final string concatenate
        if (d_column.is_null(ridx) && !col_rep.is_valid()) return 0;
        return separator_str.size_bytes() + (d_column.is_null(ridx)
                                               ? col_rep.size()
                                               : d_column.element<string_view>(ridx).size_bytes());
      },
      0,
      thrust::plus<size_type>());

    // Null/empty separator and columns doesn't produce a non-empty string
    if (bytes == 0) assert(separator_str.size_bytes() == 0);

    // Separator goes only in between elements
    return static_cast<int32_t>(bytes - separator_str.size_bytes());
  };
  auto offsets_transformer_itr = thrust::make_transform_iterator(
    thrust::make_counting_iterator<size_type>(0), offsets_transformer);
  auto offsets_column = detail::make_offsets_child_column(
    offsets_transformer_itr, offsets_transformer_itr + strings_count, stream, mr);
  auto d_results_offsets = offsets_column->view().data<int32_t>();

  // Create the chars column
  size_type bytes = thrust::device_pointer_cast(d_results_offsets)[strings_count];
  auto chars_column =
    strings::detail::create_chars_child_column(strings_count, null_count, bytes, stream, mr);

  // Fill the chars column
  auto d_results_chars = chars_column->mutable_view().data<char>();
  thrust::for_each_n(rmm::exec_policy(stream),
                     thrust::make_counting_iterator<size_type>(0),
                     strings_count,
                     [d_table,
                      num_columns,
                      d_results_offsets,
                      d_results_chars,
                      separator_col_view,
                      separator_rep,
                      col_rep] __device__(size_type ridx) {
                       // If the separator for this row is null and if there isn't a valid separator
                       // to replace, do not write anything for this row
                       if (!separator_col_view.is_valid(ridx) && !separator_rep.is_valid()) return;

                       bool const all_nulls = thrust::all_of(
                         thrust::seq, d_table.begin(), d_table.end(), [ridx](auto const& col) {
                           return col.is_null(ridx);
                         });

                       // If all column values are null and there isn't a valid column replacement,
                       // skip this row
                       if (all_nulls && !col_rep.is_valid()) return;

                       char* d_buffer      = d_results_chars + d_results_offsets[ridx];
                       bool colval_written = false;

                       // There is at least one non-null column value (it can still be empty though)
                       auto const separator_str = separator_col_view.is_valid(ridx)
                                                    ? separator_col_view.element<string_view>(ridx)
                                                    : separator_rep.value();

                       // Write out each column's entry for this row
                       for (size_type col_idx = 0; col_idx < num_columns; ++col_idx) {
                         auto const d_column = d_table.column(col_idx);
                         // If the row is null and if there is no replacement, skip it
                         if (d_column.is_null(ridx) && !col_rep.is_valid()) continue;

                         // Separator goes only in between elements
                         if (colval_written)
                           d_buffer = detail::copy_string(d_buffer, separator_str);

                         string_view const d_str = d_column.is_null(ridx)
                                                     ? col_rep.value()
                                                     : d_column.element<string_view>(ridx);
                         d_buffer                = detail::copy_string(d_buffer, d_str);
                         colval_written          = true;
                       }
                     });

  return make_strings_column(strings_count,
                             std::move(offsets_column),
                             std::move(chars_column),
                             null_count,
                             (null_count) ? std::move(valid_mask.first) : rmm::device_buffer{},
                             stream,
                             mr);
}

}  // namespace detail

// APIs

std::unique_ptr<column> concatenate(table_view const& strings_columns,
                                    string_scalar const& separator,
                                    string_scalar const& narep,
                                    rmm::mr::device_memory_resource* mr)
{
  CUDF_FUNC_RANGE();
  return detail::concatenate(strings_columns, separator, narep, rmm::cuda_stream_default, mr);
}

std::unique_ptr<column> join_strings(strings_column_view const& strings,
                                     string_scalar const& separator,
                                     string_scalar const& narep,
                                     rmm::mr::device_memory_resource* mr)
{
  CUDF_FUNC_RANGE();
  return detail::join_strings(strings, separator, narep, rmm::cuda_stream_default, mr);
}

std::unique_ptr<column> concatenate(table_view const& strings_columns,
                                    strings_column_view const& separators,
                                    string_scalar const& separator_narep,
                                    string_scalar const& col_narep,
                                    rmm::mr::device_memory_resource* mr)
{
  CUDF_FUNC_RANGE();
  return detail::concatenate(
    strings_columns, separators, separator_narep, col_narep, rmm::cuda_stream_default, mr);
}

namespace detail {
std::unique_ptr<column> concatenate(lists_column_view const& lists_strings_column,
                                    strings_column_view const& separators,
                                    string_scalar const& separator_narep,
                                    string_scalar const& string_narep,
                                    rmm::cuda_stream_view stream,
                                    rmm::mr::device_memory_resource* mr)
{
  CUDF_EXPECTS(lists_strings_column.child().type().id() == type_id::STRING,
               "The input column must be a column of lists of strings");
  CUDF_EXPECTS(lists_strings_column.size() == separators.size(),
               "Separators column should be the same size as the lists columns");

  auto const num_rows = lists_strings_column.size();
  if (num_rows == 0) { return detail::make_empty_strings_column(stream, mr); }

  auto const lists_dv_ptr    = column_device_view::create(lists_strings_column.parent(), stream);
  auto const lists_dv        = *lists_dv_ptr;
  auto const strings_col     = strings_column_view(lists_strings_column.get_sliced_child(stream));
  auto const strings_dv_ptr  = column_device_view::create(strings_col.parent(), stream);
  auto const strings_dv      = *strings_dv_ptr;
  auto const sep_dv_ptr      = column_device_view::create(separators.parent(), stream);
  auto const sep_dv          = *sep_dv_ptr;
  auto const sep_narep_dv    = get_scalar_device_view(const_cast<string_scalar&>(separator_narep));
  auto const string_narep_dv = get_scalar_device_view(const_cast<string_scalar&>(string_narep));

  // Compute sizes of strings in the output column along with their validity
  // An invalid size will be returned to indicate that the corresponding row is null
  static constexpr auto invalid_size = std::numeric_limits<size_type>::lowest();
  auto const string_size_comp_fn     = [lists_offsets = lists_strings_column.offsets_begin(),
                                    lists_dv,
                                    strings_dv,
                                    sep_dv,
                                    sep_narep_dv,
                                    string_narep_dv] __device__(size_type lidx) -> size_type {
    if (lists_dv.is_null(lidx) || (sep_dv.is_null(lidx) && !sep_narep_dv.is_valid())) {
      return invalid_size;
    }

    auto const separator_size =
      sep_dv.is_valid(lidx) ? sep_dv.element<string_view>(lidx).size_bytes() : sep_narep_dv.size();

    auto size_bytes = size_type{0};
    for (size_type str_idx = lists_offsets[lidx], idx_end = lists_offsets[lidx + 1];
         str_idx < idx_end;
         ++str_idx) {
      if (strings_dv.is_null(str_idx) && !string_narep_dv.is_valid()) {
        size_bytes = invalid_size;
        break;  // early termination: the entire list of strings will result in a null string
      }
      size_bytes += separator_size + (strings_dv.is_null(str_idx)
                                            ? string_narep_dv.size()
                                            : strings_dv.element<string_view>(str_idx).size_bytes());
    }

    // Null/empty separator and strings don't produce a non-empty string
    assert(size_bytes == invalid_size || size_bytes > separator_size ||
           (size_bytes == 0 && separator_size == 0));

    // Separator is inserted only in between strings
    return size_bytes != invalid_size ? static_cast<size_type>(size_bytes - separator_size)
                                          : invalid_size;
  };

  // Offset of the output strings
  static_assert(sizeof(offset_type) == sizeof(int32_t));
  auto offsets_column = make_numeric_column(
    data_type{type_id::INT32}, num_rows + 1, mask_state::UNALLOCATED, stream, mr);
  auto const output_offsets_ptr = offsets_column->mutable_view().begin<offset_type>();

  // Firstly, store the strings' sizes into output_str_offsets from the second element
  auto const count_it = thrust::make_counting_iterator<size_type>(0);
  CUDA_TRY(hipMemsetAsync(output_offsets_ptr, 0, sizeof(int32_t), stream.value()));
  thrust::transform(rmm::exec_policy(stream),
                    count_it,
                    count_it + num_rows,
                    output_offsets_ptr + 1,
                    string_size_comp_fn);

  // Use the strings's sizes to compute null_mask and null_count of the output strings column
  auto [null_mask, null_count] = cudf::detail::valid_if(
    count_it,
    count_it + num_rows,
    [str_sizes = output_offsets_ptr + 1] __device__(size_type idx) {
      return str_sizes[idx] != invalid_size;
    },
    stream,
    mr);

  // Build the strings's offsets from strings' sizes
  auto const iter_trans_begin = thrust::make_transform_iterator(
    output_offsets_ptr + 1,
    [] __device__(auto const size) { return size != invalid_size ? size : 0; });
  thrust::inclusive_scan(rmm::exec_policy(stream),
                         iter_trans_begin,
                         iter_trans_begin + num_rows,
                         output_offsets_ptr + 1);

  // Create the chars column to store the result strings
  auto const total_bytes = thrust::device_pointer_cast(output_offsets_ptr)[num_rows];
  auto chars_column =
    strings::detail::create_chars_child_column(num_rows, null_count, total_bytes, stream, mr);

  auto const concat_strings_fn = [lists_offsets = lists_strings_column.offsets_begin(),
                                  str_offsets   = output_offsets_ptr,
                                  output_begin  = chars_column->mutable_view().begin<char>(),
                                  strings_dv,
                                  sep_dv,
                                  sep_narep_dv,
                                  string_narep_dv] __device__(size_type out_idx) {
    if (str_offsets[out_idx + 1] == str_offsets[out_idx]) { return; }

    auto const separator =
      sep_dv.is_valid(out_idx) ? sep_dv.element<string_view>(out_idx) : sep_narep_dv.value();
    bool written    = false;
    auto output_ptr = output_begin + str_offsets[out_idx];

    for (size_type str_idx = lists_offsets[out_idx], idx_end = lists_offsets[out_idx + 1];
         str_idx < idx_end;
         ++str_idx) {
      // Separator is inserted only in between strings
      if (written) output_ptr = detail::copy_string(output_ptr, separator);
      auto const d_str = strings_dv.is_null(str_idx) ? string_narep_dv.value()
                                                     : strings_dv.element<string_view>(str_idx);
      output_ptr       = detail::copy_string(output_ptr, d_str);
      written          = true;
    }
  };

  // Finally, fill the output chars column
  thrust::for_each_n(rmm::exec_policy(stream), count_it, num_rows, concat_strings_fn);

  return make_strings_column(num_rows,
                             std::move(offsets_column),
                             std::move(chars_column),
                             null_count,
                             (null_count) ? std::move(null_mask) : rmm::device_buffer{},
                             stream,
                             mr);
}

std::unique_ptr<column> concatenate(lists_column_view const& lists_strings_column,
                                    string_scalar const& separator,
                                    string_scalar const& narep,
                                    rmm::cuda_stream_view stream,
                                    rmm::mr::device_memory_resource* mr)
{
  CUDF_EXPECTS(lists_strings_column.child().type().id() == type_id::STRING,
               "The input column must be a column of lists of strings");
  CUDF_EXPECTS(separator.is_valid(), "Parameter separator must be a valid string_scalar");

  auto const num_rows = lists_strings_column.size();
  if (num_rows == 0) { return detail::make_empty_strings_column(stream, mr); }

  auto const lists_dv_ptr    = column_device_view::create(lists_strings_column.parent(), stream);
  auto const lists_dv        = *lists_dv_ptr;
  auto const strings_col     = strings_column_view(lists_strings_column.get_sliced_child(stream));
  auto const strings_dv_ptr  = column_device_view::create(strings_col.parent(), stream);
  auto const strings_dv      = *strings_dv_ptr;
  auto const sep_dv          = get_scalar_device_view(const_cast<string_scalar&>(separator));
  auto const string_narep_dv = get_scalar_device_view(const_cast<string_scalar&>(narep));

  // Compute sizes of strings in the output column along with their validity
  // An invalid size will be returned to indicate that the corresponding row is null
  static constexpr auto invalid_size = std::numeric_limits<size_type>::lowest();
  auto const separator_size          = separator.size();
  auto const string_size_comp_fn     = [lists_offsets = lists_strings_column.offsets_begin(),
                                    lists_dv,
                                    strings_dv,
                                    separator_size,
                                    string_narep_dv] __device__(size_type lidx) -> size_type {
    if (lists_dv.is_null(lidx)) { return invalid_size; }

    auto size_bytes = size_type{0};
    for (size_type str_idx = lists_offsets[lidx], idx_end = lists_offsets[lidx + 1];
         str_idx < idx_end;
         ++str_idx) {
      if (strings_dv.is_null(str_idx) && !string_narep_dv.is_valid()) {
        size_bytes = invalid_size;
        break;  // early termination: the entire list of strings will result in a null string
      }
      size_bytes += separator_size + (strings_dv.is_null(str_idx)
                                            ? string_narep_dv.size()
                                            : strings_dv.element<string_view>(str_idx).size_bytes());
    }

    // Null/empty separator and strings don't produce a non-empty string
    assert(size_bytes == invalid_size || size_bytes > separator_size ||
           (size_bytes == 0 && separator_size == 0));

    // Separator is inserted only in between strings
    return size_bytes != invalid_size ? static_cast<size_type>(size_bytes - separator_size)
                                          : invalid_size;
  };

  // Offset of the output strings
  static_assert(sizeof(offset_type) == sizeof(int32_t));
  auto offsets_column = make_numeric_column(
    data_type{type_id::INT32}, num_rows + 1, mask_state::UNALLOCATED, stream, mr);
  auto const output_offsets_ptr = offsets_column->mutable_view().begin<offset_type>();

  // Firstly, store the strings' sizes into output_str_offsets from the second element
  auto const count_it = thrust::make_counting_iterator<size_type>(0);
  CUDA_TRY(hipMemsetAsync(output_offsets_ptr, 0, sizeof(int32_t), stream.value()));
  thrust::transform(rmm::exec_policy(stream),
                    count_it,
                    count_it + num_rows,
                    output_offsets_ptr + 1,
                    string_size_comp_fn);

  // Use the strings's sizes to compute null_mask and null_count of the output strings column
  auto [null_mask, null_count] = cudf::detail::valid_if(
    count_it,
    count_it + num_rows,
    [str_sizes = output_offsets_ptr + 1] __device__(size_type idx) {
      return str_sizes[idx] != invalid_size;
    },
    stream,
    mr);

  // Build the strings's offsets from strings' sizes
  auto const iter_trans_begin = thrust::make_transform_iterator(
    output_offsets_ptr + 1,
    [] __device__(auto const size) { return size != invalid_size ? size : 0; });
  thrust::inclusive_scan(rmm::exec_policy(stream),
                         iter_trans_begin,
                         iter_trans_begin + num_rows,
                         output_offsets_ptr + 1);

  // Create the chars column to store the result strings
  auto const total_bytes = thrust::device_pointer_cast(output_offsets_ptr)[num_rows];
  auto chars_column =
    strings::detail::create_chars_child_column(num_rows, null_count, total_bytes, stream, mr);

  auto const concat_strings_fn = [lists_offsets = lists_strings_column.offsets_begin(),
                                  str_offsets   = output_offsets_ptr,
                                  output_begin  = chars_column->mutable_view().begin<char>(),
                                  strings_dv,
                                  sep_dv,
                                  string_narep_dv] __device__(size_type out_idx) {
    if (str_offsets[out_idx + 1] == str_offsets[out_idx]) { return; }

    auto const separator = sep_dv.value();
    bool written         = false;
    auto output_ptr      = output_begin + str_offsets[out_idx];

    for (size_type str_idx = lists_offsets[out_idx], idx_end = lists_offsets[out_idx + 1];
         str_idx < idx_end;
         ++str_idx) {
      // Separator is inserted only in between strings
      if (written) output_ptr = detail::copy_string(output_ptr, separator);
      auto const d_str = strings_dv.is_null(str_idx) ? string_narep_dv.value()
                                                     : strings_dv.element<string_view>(str_idx);
      output_ptr       = detail::copy_string(output_ptr, d_str);
      written          = true;
    }
  };

  // Finally, fill the output chars column
  thrust::for_each_n(rmm::exec_policy(stream), count_it, num_rows, concat_strings_fn);

  return make_strings_column(num_rows,
                             std::move(offsets_column),
                             std::move(chars_column),
                             null_count,
                             (null_count) ? std::move(null_mask) : rmm::device_buffer{},
                             stream,
                             mr);
}
}  // namespace detail

std::unique_ptr<column> concatenate(lists_column_view const& lists_strings_column,
                                    strings_column_view const& separators,
                                    string_scalar const& separator_narep,
                                    string_scalar const& string_narep,
                                    rmm::mr::device_memory_resource* mr)
{
  CUDF_FUNC_RANGE();
  return detail::concatenate(
    lists_strings_column, separators, separator_narep, string_narep, rmm::cuda_stream_default, mr);
}
std::unique_ptr<column> concatenate(lists_column_view const& lists_strings_column,
                                    string_scalar const& separator,
                                    string_scalar const& narep,
                                    rmm::mr::device_memory_resource* mr)
{
  CUDF_FUNC_RANGE();
  return detail::concatenate(lists_strings_column, separator, narep, rmm::cuda_stream_default, mr);
}

}  // namespace strings
}  // namespace cudf
