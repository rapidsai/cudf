/*
 * Copyright (c) 2019-2020, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <cudf/column/column_device_view.cuh>
#include <cudf/column/column_factories.hpp>
#include <cudf/detail/nvtx/ranges.hpp>
#include <cudf/scalar/scalar_device_view.cuh>
#include <cudf/strings/detail/utilities.hpp>
#include <cudf/strings/string_view.cuh>
#include <cudf/strings/strings_column_view.hpp>
#include <cudf/strings/substring.hpp>
#include <cudf/utilities/traits.hpp>
#include <cudf/utilities/type_dispatcher.hpp>
#include <strings/utilities.cuh>

namespace {

/**
 * @brief Used as template parameter to divide size calculation from
 * the actual string operation within a function.
 *
 * Useful when most of the logic is identical for both passes.
 */
enum TwoPass {
  SizeOnly = 0,  ///< calculate the size only
  ExecuteOp      ///< run the string operation
};

}  // namespace

namespace cudf {
namespace strings {
namespace detail {
namespace {

/**
 * @brief Function logic for the substring API.
 *
 * This will perform a substring operation on each string
 * using the provided start, stop, and step parameters.
 */
struct substring_fn {
  const column_device_view d_column;
  numeric_scalar_device_view<size_type> d_start, d_stop, d_step;
  const int32_t* d_offsets{};
  char* d_chars{};

  __device__ cudf::size_type operator()(size_type idx) {
    if (d_column.is_null(idx)) return 0;  // null string
    string_view d_str = d_column.template element<string_view>(idx);
    auto const length = d_str.length();
    if (length == 0) return 0;  // empty string
    size_type const step = d_step.is_valid() ? d_step.value() : 1;
    auto const begin     = [&] {  // always inclusive
      // when invalid, default depends on step
      if (!d_start.is_valid()) return (step > 0) ? d_str.begin() : (d_str.end() - 1);
      // normal positive position logic
      auto start = d_start.value();
      if (start >= 0) {
        if (start < length) return d_str.begin() + start;
        return d_str.end() + (step < 0 ? -1 : 0);
      }
      // handle negative position here
      auto adjust = length + start;
      if (adjust >= 0) return d_str.begin() + adjust;
      return d_str.begin() + (step < 0 ? -1 : 0);
    }();
    auto const end = [&] {  // always exclusive
      // when invalid, default depends on step
      if (!d_stop.is_valid()) return step > 0 ? d_str.end() : (d_str.begin() - 1);
      // normal positive position logic
      auto stop = d_stop.value();
      if (stop >= 0) return (stop < length) ? (d_str.begin() + stop) : d_str.end();
      // handle negative position here
      auto adjust = length + stop;
      return d_str.begin() + (adjust >= 0 ? adjust : -1);
    }();

    size_type bytes = 0;
    char* d_buffer  = d_chars ? d_chars + d_offsets[idx] : nullptr;
    auto itr        = begin;
    while (step > 0 ? itr < end : end < itr) {
      bytes += bytes_in_char_utf8(*itr);
      if (d_buffer) d_buffer += from_char_utf8(*itr, d_buffer);
      itr += step;
    }
    return bytes;
  }
};

}  // namespace

//
std::unique_ptr<column> slice_strings(
  strings_column_view const& strings,
  numeric_scalar<size_type> const& start = numeric_scalar<size_type>(0, false),
  numeric_scalar<size_type> const& stop  = numeric_scalar<size_type>(0, false),
  numeric_scalar<size_type> const& step  = numeric_scalar<size_type>(1),
  rmm::mr::device_memory_resource* mr    = rmm::mr::get_default_resource(),
  hipStream_t stream                    = 0) {
  size_type strings_count = strings.size();
  if (strings_count == 0) return make_empty_strings_column(mr, stream);

  if (step.is_valid()) CUDF_EXPECTS(step.value(stream) != 0, "Step parameter must not be 0");

  auto strings_column = column_device_view::create(strings.parent(), stream);
  auto d_column       = *strings_column;
  auto d_start        = get_scalar_device_view(const_cast<numeric_scalar<size_type>&>(start));
  auto d_stop         = get_scalar_device_view(const_cast<numeric_scalar<size_type>&>(stop));
  auto d_step         = get_scalar_device_view(const_cast<numeric_scalar<size_type>&>(step));

  // copy the null mask
  rmm::device_buffer null_mask = copy_bitmask(strings.parent(), stream, mr);

  // build offsets column
  auto offsets_transformer_itr = thrust::make_transform_iterator(
    thrust::make_counting_iterator<int32_t>(0), substring_fn{d_column, d_start, d_stop, d_step});
  auto offsets_column = make_offsets_child_column(
    offsets_transformer_itr, offsets_transformer_itr + strings_count, mr, stream);
  auto d_new_offsets = offsets_column->view().data<int32_t>();

  // build chars column
  size_type bytes   = thrust::device_pointer_cast(d_new_offsets)[strings_count];
  auto chars_column = strings::detail::create_chars_child_column(
    strings_count, strings.null_count(), bytes, mr, stream);
  auto d_chars = chars_column->mutable_view().data<char>();
  thrust::for_each_n(rmm::exec_policy(stream)->on(stream),
                     thrust::make_counting_iterator<size_type>(0),
                     strings_count,
                     substring_fn{d_column, d_start, d_stop, d_step, d_new_offsets, d_chars});
  //
  return make_strings_column(strings_count,
                             std::move(offsets_column),
                             std::move(chars_column),
                             strings.null_count(),
                             std::move(null_mask),
                             stream,
                             mr);
}

}  // namespace detail

// external API

std::unique_ptr<column> slice_strings(strings_column_view const& strings,
                                      numeric_scalar<size_type> const& start,
                                      numeric_scalar<size_type> const& stop,
                                      numeric_scalar<size_type> const& step,
                                      rmm::mr::device_memory_resource* mr) {
  CUDF_FUNC_RANGE();
  return detail::slice_strings(strings, start, stop, step, mr);
}

namespace detail {
namespace {

template <typename PositionType, TwoPass Pass = SizeOnly>
struct substring_from_fn {
  const column_device_view d_column;
  const PositionType* starts;
  const PositionType* stops;
  const int32_t* d_offsets{};
  char* d_chars{};

  /**
     * @brief Function logic for substring_from API.
     * This does both calculate and the execute based on template parameter.
     */
  __device__ size_type operator()(size_type idx) {
    if (d_column.is_null(idx)) return 0;  // null string
    string_view d_str = d_column.template element<string_view>(idx);
    size_type length  = d_str.length();
    size_type start   = static_cast<size_type>(starts[idx]);
    if (start >= length) return 0;  // empty string
    size_type stop       = static_cast<size_type>(stops[idx]);
    size_type end        = (((stop < 0) || (stop > length)) ? length : stop);
    string_view d_substr = d_str.substr(start, end - start);
    if (Pass == SizeOnly)
      return d_substr.size_bytes();
    else {
      memcpy(d_chars + d_offsets[idx], d_substr.data(), d_substr.size_bytes());
      return 0;
    }
  }
};

/**
 * Called by the type-dispatcher for resolving the position columns
 * (starts_column and stops_column) to actual types.
 */
struct dispatch_substring_from_fn {
  /**
     * @brief Returns strings column with substrings based on the ranges in the
     * individual starts and stops column position values.
     */
  template <typename PositionType,
            std::enable_if_t<std::is_integral<PositionType>::value>* = nullptr>
  std::unique_ptr<column> operator()(strings_column_view const& strings,
                                     column_view const& starts_column,
                                     column_view const& stops_column,
                                     rmm::mr::device_memory_resource* mr,
                                     hipStream_t stream) const {
    const PositionType* starts = starts_column.data<PositionType>();
    const PositionType* stops  = stops_column.data<PositionType>();

    auto strings_count  = strings.size();
    auto strings_column = column_device_view::create(strings.parent(), stream);
    auto d_column       = *strings_column;

    // copy the null mask
    rmm::device_buffer null_mask;
    size_type null_count = strings.null_count();
    if (d_column.nullable())
      null_mask = rmm::device_buffer(
        d_column.null_mask(), cudf::bitmask_allocation_size_bytes(strings_count), stream, mr);
    // build offsets column
    auto offsets_transformer_itr =
      thrust::make_transform_iterator(thrust::make_counting_iterator<PositionType>(0),
                                      substring_from_fn<PositionType>{d_column, starts, stops});
    auto offsets_column = cudf::strings::detail::make_offsets_child_column(
      offsets_transformer_itr, offsets_transformer_itr + strings_count, mr, stream);
    auto offsets_view  = offsets_column->view();
    auto d_new_offsets = offsets_view.template data<int32_t>();

    // build chars column
    cudf::size_type bytes = thrust::device_pointer_cast(d_new_offsets)[strings_count];
    auto chars_column     = cudf::strings::detail::create_chars_child_column(
      strings_count, null_count, bytes, mr, stream);
    auto chars_view = chars_column->mutable_view();
    auto d_chars    = chars_view.template data<char>();
    thrust::for_each_n(
      rmm::exec_policy(stream)->on(stream),
      thrust::make_counting_iterator<cudf::size_type>(0),
      strings_count,
      substring_from_fn<PositionType, ExecuteOp>{d_column, starts, stops, d_new_offsets, d_chars});
    //
    return make_strings_column(strings_count,
                               std::move(offsets_column),
                               std::move(chars_column),
                               null_count,
                               std::move(null_mask),
                               stream,
                               mr);
  }
  //
  template <typename PositionType,
            std::enable_if_t<not std::is_integral<PositionType>::value>* = nullptr>
  std::unique_ptr<column> operator()(strings_column_view const&,
                                     column_view const&,
                                     column_view const&,
                                     rmm::mr::device_memory_resource*,
                                     hipStream_t) const {
    CUDF_FAIL("Positions values must be an integral type.");
  }
};

template <>
std::unique_ptr<column> dispatch_substring_from_fn::operator()<bool>(
  strings_column_view const&,
  column_view const&,
  column_view const&,
  rmm::mr::device_memory_resource*,
  hipStream_t) const {
  CUDF_FAIL("Positions values must not be bool type.");
}

}  // namespace

//
std::unique_ptr<column> slice_strings(
  strings_column_view const& strings,
  column_view const& starts_column,
  column_view const& stops_column,
  rmm::mr::device_memory_resource* mr = rmm::mr::get_default_resource(),
  hipStream_t stream                 = 0) {
  size_type strings_count = strings.size();
  if (strings_count == 0) return make_empty_strings_column(mr, stream);
  CUDF_EXPECTS(starts_column.size() == strings_count,
               "Parameter starts must have the same number of rows as strings.");
  CUDF_EXPECTS(stops_column.size() == strings_count,
               "Parameter stops must have the same number of rows as strings.");
  CUDF_EXPECTS(starts_column.type() == stops_column.type(),
               "Parameters starts and stops must be of the same type.");
  CUDF_EXPECTS(starts_column.null_count() == 0, "Parameter starts must not contain nulls.");
  CUDF_EXPECTS(stops_column.null_count() == 0, "Parameter stops must not contain nulls.");

  // perhaps another candidate for index-normalizer
  return cudf::experimental::type_dispatcher(starts_column.type(),
                                             dispatch_substring_from_fn{},
                                             strings,
                                             starts_column,
                                             stops_column,
                                             mr,
                                             stream);
}

}  // namespace detail

// external API

std::unique_ptr<column> slice_strings(strings_column_view const& strings,
                                      column_view const& starts_column,
                                      column_view const& stops_column,
                                      rmm::mr::device_memory_resource* mr) {
  CUDF_FUNC_RANGE();
  return detail::slice_strings(strings, starts_column, stops_column, mr);
}

}  // namespace strings
}  // namespace cudf
