#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019-2020, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <strings/split/split_utils.cuh>

#include <cudf/column/column.hpp>
#include <cudf/column/column_device_view.cuh>
#include <cudf/column/column_factories.hpp>
#include <cudf/detail/null_mask.hpp>
#include <cudf/detail/nvtx/ranges.hpp>
#include <cudf/strings/detail/strings_column_factories.cuh>
#include <cudf/strings/detail/utilities.hpp>
#include <cudf/strings/split/split.hpp>
#include <cudf/strings/string_view.cuh>
#include <cudf/strings/strings_column_view.hpp>
#include <cudf/utilities/error.hpp>

#include <rmm/cuda_stream_view.hpp>

#include <thrust/binary_search.h>  // upper_bound()
#include <thrust/copy.h>           // copy_if()
#include <thrust/count.h>          // count_if()
#include <thrust/extrema.h>        // max()
#include <thrust/transform.h>      // transform()

namespace cudf {
namespace strings {
namespace detail {

using string_index_pair = thrust::pair<const char*, size_type>;

namespace {

/**
 * @brief Base class for delimiter-based tokenizers.
 *
 * These are common methods used by both split and rsplit tokenizer functors.
 */
struct base_split_tokenizer {
  __device__ const char* get_base_ptr() const
  {
    return d_strings.child(strings_column_view::chars_column_index).data<char>();
  }

  __device__ string_view const get_string(size_type idx) const
  {
    return d_strings.element<string_view>(idx);
  }

  __device__ bool is_valid(size_type idx) const { return d_strings.is_valid(idx); }

  /**
   * @brief Initialize token elements for all strings.
   *
   * The process_tokens() only handles creating tokens for strings that contain
   * delimiters. This function will initialize the output tokens for all
   * strings by assigning null entries for null and empty strings and the
   * string itself for strings with no delimiters.
   *
   * The tokens are placed in output order so that all tokens for each output
   * column are stored consecutively in `d_all_tokens`.
   *
   * @param idx Index of string in column
   * @param column_count Number of columns in output
   * @param d_all_tokens Tokens vector for all strings
   */
  __device__ void init_tokens(size_type idx,
                              size_type column_count,
                              string_index_pair* d_all_tokens) const
  {
    auto d_tokens = d_all_tokens + idx;
    if (is_valid(idx)) {
      auto d_str = get_string(idx);
      *d_tokens  = string_index_pair{d_str.data(), d_str.size_bytes()};
      --column_count;
      d_tokens += d_strings.size();
    }
    // this is like fill() but output needs to be strided
    for (size_type col = 0; col < column_count; ++col)
      d_tokens[d_strings.size() * col] = string_index_pair{nullptr, 0};
  }

  base_split_tokenizer(column_device_view const& d_strings,
                       string_view const& d_delimiter,
                       size_type max_tokens)
    : d_strings(d_strings), d_delimiter(d_delimiter), max_tokens(max_tokens)
  {
  }

 protected:
  column_device_view const d_strings;  // strings to split
  string_view const d_delimiter;       // delimiter for split
  size_type max_tokens;
};

/**
 * @brief The tokenizer functions for split().
 *
 * The methods here count delimiters, tokens, and output token elements
 * for each string in a strings column.
 */
struct split_tokenizer_fn : base_split_tokenizer {
  /**
   * @brief This will create tokens around each delimiter honoring the string boundaries
   * in which the delimiter resides.
   *
   * Each token is placed in `d_all_tokens` so they align consecutively
   * with other tokens for the same output column.
   * That is, `d_tokens[col * strings_count + string_index]` is the token at column `col`
   * for string at `string_index`.
   *
   * @param idx Index of the delimiter in the chars column
   * @param column_count Number of output columns
   * @param d_token_counts Token counts for each string
   * @param d_positions The beginning byte position of each delimiter
   * @param positions_count Number of delimiters
   * @param d_indexes Indices of the strings for each delimiter
   * @param d_all_tokens All output tokens for the strings column
   */
  __device__ void process_tokens(size_type idx,
                                 size_type column_count,
                                 size_type const* d_token_counts,
                                 size_type const* d_positions,
                                 size_type positions_count,
                                 size_type const* d_indexes,
                                 string_index_pair* d_all_tokens) const
  {
    size_type str_idx = d_indexes[idx];
    if ((idx > 0) && d_indexes[idx - 1] == str_idx)
      return;   // the first delimiter for the string rules them all
    --str_idx;  // all of these are off by 1 from the upper_bound call
    size_type token_count      = d_token_counts[str_idx];  // max_tokens already included
    const char* const base_ptr = get_base_ptr();  // d_positions values are based on this ptr
    // this string's tokens output
    auto d_tokens = d_all_tokens + str_idx;
    // this string
    const string_view d_str       = get_string(str_idx);
    const char* str_ptr           = d_str.data();                  // beginning of the string
    const char* const str_end_ptr = str_ptr + d_str.size_bytes();  // end of the string
    // build the index-pair of each token for this string
    for (size_type col = 0; col < token_count; ++col) {
      auto next_delim = ((idx + col) < positions_count)  // boundary check for delims in last string
                          ? (base_ptr + d_positions[idx + col])  // start of next delimiter
                          : str_end_ptr;                         // or end of this string
      auto eptr = (next_delim < str_end_ptr)      // make sure delimiter is inside this string
                      && (col + 1 < token_count)  // and this is not the last token
                    ? next_delim
                    : str_end_ptr;
      // store the token into the output vector
      d_tokens[col * d_strings.size()] =
        string_index_pair{str_ptr, static_cast<size_type>(eptr - str_ptr)};
      // point past this delimiter
      str_ptr = eptr + d_delimiter.size_bytes();
    }
  }

  /**
   * @brief Returns `true` if the byte at `idx` is the start of the delimiter.
   *
   * @param idx Index of a byte in the chars column.
   * @param d_offsets Offsets values to locate the chars ranges.
   * @param chars_bytes Total number of characters to process.
   * @return true if delimiter is found starting at position `idx`
   */
  __device__ bool is_delimiter(size_type idx,  // chars index
                               int32_t const* d_offsets,
                               size_type chars_bytes) const
  {
    auto d_chars = get_base_ptr() + d_offsets[0];
    if (idx + d_delimiter.size_bytes() > chars_bytes) return false;
    return d_delimiter.compare(d_chars + idx, d_delimiter.size_bytes()) == 0;
  }

  /**
   * @brief This counts the tokens for strings that contain delimiters.
   *
   * @param idx Index of a delimiter
   * @param d_positions Start positions of all the delimiters
   * @param positions_count The number of delimiters
   * @param d_indexes Indices of the strings for each delimiter
   * @param d_counts The token counts for all the strings
   */
  __device__ void count_tokens(size_type idx,  // delimiter index
                               size_type const* d_positions,
                               size_type positions_count,
                               size_type const* d_indexes,
                               size_type* d_counts) const
  {
    size_type str_idx = d_indexes[idx];
    if ((idx > 0) && d_indexes[idx - 1] == str_idx)
      return;  // first delimiter found handles all of them for this string
    auto const delim_length    = d_delimiter.size_bytes();
    string_view const d_str    = get_string(str_idx - 1);
    const char* const base_ptr = get_base_ptr();
    size_type delim_count      = 0;  // re-count delimiters to compute the token-count
    size_type last_pos         = d_positions[idx] - delim_length;
    while ((idx < positions_count) && (d_indexes[idx] == str_idx)) {
      // make sure the whole delimiter is inside the string before counting it
      auto d_pos = d_positions[idx];
      if (((base_ptr + d_pos + delim_length - 1) < (d_str.data() + d_str.size_bytes())) &&
          ((d_pos - last_pos) >= delim_length)) {
        ++delim_count;     // only count if the delimiter fits
        last_pos = d_pos;  // overlapping delimiters are ignored too
      }
      ++idx;
    }
    // the number of tokens is delim_count+1 but capped to max_tokens
    d_counts[str_idx - 1] =
      ((max_tokens > 0) && (delim_count + 1 > max_tokens)) ? max_tokens : delim_count + 1;
  }

  split_tokenizer_fn(column_device_view const& d_strings,
                     string_view const& d_delimiter,
                     size_type max_tokens)
    : base_split_tokenizer(d_strings, d_delimiter, max_tokens)
  {
  }
};

/**
 * @brief The tokenizer functions for split().
 *
 * The methods here count delimiters, tokens, and output token elements
 * for each string in a strings column.
 *
 * Same as split_tokenizer_fn except tokens are counted from the end of each string.
 */
struct rsplit_tokenizer_fn : base_split_tokenizer {
  /**
   * @brief This will create tokens around each delimiter honoring the string boundaries
   * in which the delimiter resides.
   *
   * The tokens are processed from the end of each string so the `max_tokens`
   * is honored correctly.
   *
   * Each token is placed in `d_all_tokens` so they align consecutively
   * with other tokens for the same output column.
   * That is, `d_tokens[col * strings_count + string_index]` is the token at column `col`
   * for string at `string_index`.
   *
   * @param idx Index of the delimiter in the chars column
   * @param column_count Number of output columns
   * @param d_token_counts Token counts for each string
   * @param d_positions The ending byte position of each delimiter
   * @param positions_count Number of delimiters
   * @param d_indexes Indices of the strings for each delimiter
   * @param d_all_tokens All output tokens for the strings column
   */
  __device__ void process_tokens(size_type idx,                    // delimiter position index
                                 size_type column_count,           // number of output columns
                                 size_type const* d_token_counts,  // token counts for each string
                                 size_type const* d_positions,     // end of each delimiter
                                 size_type positions_count,        // total number of delimiters
                                 size_type const* d_indexes,  // string indices for each delimiter
                                 string_index_pair* d_all_tokens) const
  {
    size_type str_idx = d_indexes[idx];
    if ((idx + 1 < positions_count) && d_indexes[idx + 1] == str_idx)
      return;   // the last delimiter for the string rules them all
    --str_idx;  // all of these are off by 1 from the upper_bound call
    size_type token_count      = d_token_counts[str_idx];  // max_tokens already included
    const char* const base_ptr = get_base_ptr();  // d_positions values are based on this ptr
    // this string's tokens output
    auto d_tokens = d_all_tokens + str_idx;
    // this string
    const string_view d_str         = get_string(str_idx);
    const char* const str_begin_ptr = d_str.data();  // beginning of the string
    const char* str_ptr             = str_begin_ptr + d_str.size_bytes();  // end of the string
    // build the index-pair of each token for this string
    for (size_type col = 0; col < token_count; ++col) {
      auto prev_delim = (idx >= col)  // boundary check for delims in first string
                          ? (base_ptr + d_positions[idx - col] + 1)  // end of prev delimiter
                          : str_begin_ptr;                           // or the start of this string
      auto sptr = (prev_delim > str_begin_ptr)    // make sure delimiter is inside the string
                      && (col + 1 < token_count)  // and this is not the last token
                    ? prev_delim
                    : str_begin_ptr;
      // store the token into the output -- building the array backwards
      d_tokens[d_strings.size() * (token_count - 1 - col)] =
        string_index_pair{sptr, static_cast<size_type>(str_ptr - sptr)};
      str_ptr = sptr - d_delimiter.size_bytes();  // get ready for the next prev token
    }
  }

  /**
   * @brief Returns `true` if the byte at `idx` is the end of the delimiter.
   *
   * @param idx Index of a byte in the chars column.
   * @param d_offsets Offsets values to locate the chars ranges.
   * @param chars_bytes Total number of characters to process.
   * @return true if delimiter is found ending at position `idx`
   */
  __device__ bool is_delimiter(size_type idx, int32_t const* d_offsets, size_type chars_bytes) const
  {
    auto delim_length = d_delimiter.size_bytes();
    if (idx < delim_length - 1) return false;
    auto d_chars = get_base_ptr() + d_offsets[0];
    return d_delimiter.compare(d_chars + idx - (delim_length - 1), delim_length) == 0;
  }

  /**
   * @brief This counts the tokens for strings that contain delimiters.
   *
   * Token counting starts at the end of the string to honor the `max_tokens`
   * appropriately.
   *
   * @param idx Index of a delimiter
   * @param d_positions End positions of all the delimiters
   * @param positions_count The number of delimiters
   * @param d_indexes Indices of the strings for each delimiter
   * @param d_counts The token counts for all the strings
   */
  __device__ void count_tokens(size_type idx,
                               size_type const* d_positions,
                               size_type positions_count,
                               size_type const* d_indexes,
                               size_type* d_counts) const
  {
    size_type str_idx = d_indexes[idx];  // 1-based string index created by upper_bound()
    if ((idx > 0) && d_indexes[idx - 1] == str_idx)
      return;  // first delimiter found handles all of them for this string
    auto const delim_length    = d_delimiter.size_bytes();
    const string_view d_str    = get_string(str_idx - 1);  // -1 for 0-based index
    const char* const base_ptr = get_base_ptr();
    size_type delim_count      = 0;
    size_type last_pos         = d_positions[idx] - delim_length;
    while ((idx < positions_count) && (d_indexes[idx] == str_idx)) {
      // make sure the whole delimiter is inside the string before counting it
      auto d_pos = d_positions[idx];
      if (((base_ptr + d_pos + 1 - delim_length) >= d_str.data()) &&
          ((d_pos - last_pos) >= delim_length)) {
        ++delim_count;     // only count if the delimiter fits
        last_pos = d_pos;  // overlapping delimiters are also ignored
      }
      ++idx;
    }
    // the number of tokens is delim_count+1 but capped to max_tokens
    d_counts[str_idx - 1] =
      ((max_tokens > 0) && (delim_count + 1 > max_tokens)) ? max_tokens : delim_count + 1;
  }

  rsplit_tokenizer_fn(column_device_view const& d_strings,
                      string_view const& d_delimiter,
                      size_type max_tokens)
    : base_split_tokenizer(d_strings, d_delimiter, max_tokens)
  {
  }
};

/**
 * @brief Generic split function called by split() and rsplit().
 *
 * This function will first count the number of delimiters in the entire strings
 * column. Next it records the position of all the delimiters. These positions
 * are used for the remainder of the code to build string_index_pair elements
 * for each output column.
 *
 * The number of tokens for each string is computed by analyzing the delimiter
 * position values and mapping them to each string.
 * The number of output columns is determined by the string with the most tokens.
 * Next the `string_index_pairs` for the entire column are created using the
 * delimiter positions and their string indices vector.
 *
 * Finally, each column is built by creating a vector of tokens (`string_index_pairs`)
 * according to their position in each string. The first token from each string goes
 * into the first output column, the 2nd token from each string goes into the 2nd
 * output column, etc.
 *
 * Output should be comparable to Pandas `split()` with `expand=True` but the
 * rows/columns are transposed.
 *
 * ```
 *   import pandas as pd
 *   pd_series = pd.Series(['', None, 'a_b', '_a_b_', '__aa__bb__', '_a__bbb___c', '_aa_b__ccc__'])
 *   print(pd_series.str.split(pat='_', expand=True))
 *            0     1     2     3     4     5     6
 *      0    ''  None  None  None  None  None  None
 *      1  None  None  None  None  None  None  None
 *      2     a     b  None  None  None  None  None
 *      3    ''     a     b    ''  None  None  None
 *      4    ''    ''    aa    ''    bb    ''    ''
 *      5    ''     a    ''   bbb    ''    ''     c
 *      6    ''    aa     b    ''   ccc    ''    ''
 *
 *   print(pd_series.str.split(pat='_', n=1, expand=True))
 *            0            1
 *      0    ''         None
 *      1  None         None
 *      2     a            b
 *      3    ''         a_b_
 *      4    ''    _aa__bb__
 *      5    ''   a__bbb___c
 *      6    ''  aa_b__ccc__
 *
 *   print(pd_series.str.split(pat='_', n=2, expand=True))
 *            0     1         2
 *      0    ''  None      None
 *      1  None  None      None
 *      2     a     b      None
 *      3    ''     a        b_
 *      4    ''        aa__bb__
 *      5    ''     a  _bbb___c
 *      6    ''    aa  b__ccc__
 * ```
 *
 * @tparam Tokenizer provides unique functions for split/rsplit.
 * @param strings_column The strings to split
 * @param tokenizer Tokenizer for counting and producing tokens
 * @return table of columns for the output of the split
 */
template <typename Tokenizer>
std::unique_ptr<table> split_fn(strings_column_view const& strings_column,
                                Tokenizer tokenizer,
                                rmm::cuda_stream_view stream,
                                rmm::mr::device_memory_resource* mr)
{
  std::vector<std::unique_ptr<column>> results;
  auto strings_count = strings_column.size();
  if (strings_count == 0) {
    results.push_back(make_empty_strings_column(stream, mr));
    return std::make_unique<table>(std::move(results));
  }

  auto execpol   = rmm::exec_policy(stream);
  auto d_offsets = strings_column.offsets().data<int32_t>();
  d_offsets += strings_column.offset();  // nvbug-2808421 : do not combine with the previous line
  auto chars_bytes = thrust::device_pointer_cast(d_offsets)[strings_count] -
                     thrust::device_pointer_cast(d_offsets)[0];

  // count the number of delimiters in the entire column
  size_type delimiter_count =
    thrust::count_if(execpol->on(stream.value()),
                     thrust::make_counting_iterator<size_type>(0),
                     thrust::make_counting_iterator<size_type>(chars_bytes),
                     [tokenizer, d_offsets, chars_bytes] __device__(size_type idx) {
                       return tokenizer.is_delimiter(idx, d_offsets, chars_bytes);
                     });

  // create vector of every delimiter position in the chars column
  rmm::device_vector<size_type> delimiter_positions(delimiter_count);
  auto d_positions = delimiter_positions.data().get();
  auto copy_end    = thrust::copy_if(execpol->on(stream.value()),
                                  thrust::make_counting_iterator<size_type>(0),
                                  thrust::make_counting_iterator<size_type>(chars_bytes),
                                  delimiter_positions.begin(),
                                  [tokenizer, d_offsets, chars_bytes] __device__(size_type idx) {
                                    return tokenizer.is_delimiter(idx, d_offsets, chars_bytes);
                                  });

  // create vector of string indices for each delimiter
  rmm::device_vector<size_type> string_indices(delimiter_count);  // these will be strings that
  auto d_string_indices = string_indices.data().get();            // only contain delimiters
  thrust::upper_bound(execpol->on(stream.value()),
                      d_offsets,
                      d_offsets + strings_count,
                      delimiter_positions.begin(),
                      copy_end,
                      string_indices.begin());

  // compute the number of tokens per string
  rmm::device_vector<size_type> token_counts(strings_count);
  auto d_token_counts = token_counts.data().get();
  // first, initialize token counts for strings without delimiters in them
  thrust::transform(execpol->on(stream.value()),
                    thrust::make_counting_iterator<size_type>(0),
                    thrust::make_counting_iterator<size_type>(strings_count),
                    d_token_counts,
                    [tokenizer] __device__(size_type idx) {
                      // null are 0, all others 1
                      return static_cast<size_type>(tokenizer.is_valid(idx));
                    });
  // now compute the number of tokens in each string
  thrust::for_each_n(
    execpol->on(stream.value()),
    thrust::make_counting_iterator<size_type>(0),
    delimiter_count,
    [tokenizer, d_positions, delimiter_count, d_string_indices, d_token_counts] __device__(
      size_type idx) {
      tokenizer.count_tokens(idx, d_positions, delimiter_count, d_string_indices, d_token_counts);
    });

  // the columns_count is the maximum number of tokens for any string
  size_type columns_count =
    *thrust::max_element(execpol->on(stream.value()), token_counts.begin(), token_counts.end());
  // boundary case: if no columns, return one null column (custrings issue #119)
  if (columns_count == 0) {
    results.push_back(std::make_unique<column>(
      data_type{type_id::STRING},
      strings_count,
      rmm::device_buffer{0, stream, mr},  // no data
      cudf::detail::create_null_mask(strings_count, mask_state::ALL_NULL, stream, mr),
      strings_count));
  }

  // create working area to hold all token positions
  rmm::device_vector<string_index_pair> tokens(columns_count * strings_count);
  string_index_pair* d_tokens = tokens.data().get();
  // initialize the token positions
  // -- accounts for nulls, empty, and strings with no delimiter in them
  thrust::for_each_n(execpol->on(stream.value()),
                     thrust::make_counting_iterator<size_type>(0),
                     strings_count,
                     [tokenizer, columns_count, d_tokens] __device__(size_type idx) {
                       tokenizer.init_tokens(idx, columns_count, d_tokens);
                     });

  // get the positions for every token using the delimiter positions
  thrust::for_each_n(execpol->on(stream.value()),
                     thrust::make_counting_iterator<size_type>(0),
                     delimiter_count,
                     [tokenizer,
                      columns_count,
                      d_token_counts,
                      d_positions,
                      delimiter_count,
                      d_string_indices,
                      d_tokens] __device__(size_type idx) {
                       tokenizer.process_tokens(idx,
                                                columns_count,
                                                d_token_counts,
                                                d_positions,
                                                delimiter_count,
                                                d_string_indices,
                                                d_tokens);
                     });

  // Create each column.
  // - Each pair points to the strings for that column for each row.
  // - Create the strings column from the vector using the strings factory.
  for (size_type col = 0; col < columns_count; ++col) {
    auto column_tokens = d_tokens + (col * strings_count);
    results.emplace_back(
      make_strings_column(column_tokens, column_tokens + strings_count, stream, mr));
  }
  return std::make_unique<table>(std::move(results));
}

/**
 * @brief Base class for whitespace tokenizers.
 *
 * These are common methods used by both split and rsplit tokenizer functors.
 */
struct base_whitespace_split_tokenizer {
  // count the tokens only between non-whitespace characters
  __device__ size_type count_tokens(size_type idx) const
  {
    if (d_strings.is_null(idx)) return 0;
    const string_view d_str = d_strings.element<string_view>(idx);
    size_type token_count   = 0;
    // run of whitespace is considered a single delimiter
    bool spaces = true;
    auto itr    = d_str.begin();
    while (itr != d_str.end()) {
      char_utf8 ch = *itr;
      if (spaces == (ch <= ' '))
        itr++;
      else {
        token_count += static_cast<size_type>(spaces);
        spaces = !spaces;
      }
    }
    if (max_tokens && (token_count > max_tokens)) token_count = max_tokens;
    if (token_count == 0) token_count = 1;  // always at least 1 token
    return token_count;
  }

  base_whitespace_split_tokenizer(column_device_view const& d_strings, size_type max_tokens)
    : d_strings(d_strings), max_tokens(max_tokens)
  {
  }

 protected:
  column_device_view const d_strings;
  size_type max_tokens;  // maximum number of tokens
};

/**
 * @brief The tokenizer functions for split() with whitespace.
 *
 * The whitespace tokenizer has no delimiter and handles one or more
 * consecutive whitespace characters as a single delimiter.
 */
struct whitespace_split_tokenizer_fn : base_whitespace_split_tokenizer {
  /**
   * @brief This will create tokens around each runs of whitespace characters.
   *
   * Each token is placed in `d_all_tokens` so they align consecutively
   * with other tokens for the same output column.
   * That is, `d_tokens[col * strings_count + string_index]` is the token at column `col`
   * for string at `string_index`.
   *
   * @param idx Index of the string to process
   * @param column_count Number of output columns
   * @param d_token_counts Token counts for each string
   * @param d_all_tokens All output tokens for the strings column
   */
  __device__ void process_tokens(size_type idx,
                                 size_type column_count,
                                 size_type const* d_token_counts,
                                 string_index_pair* d_all_tokens) const
  {
    string_index_pair* d_tokens = d_all_tokens + idx;
    if (d_strings.is_null(idx)) return;
    string_view const d_str = d_strings.element<cudf::string_view>(idx);
    if (d_str.empty()) return;
    whitespace_string_tokenizer tokenizer(d_str);
    size_type token_count = d_token_counts[idx];
    size_type token_idx   = 0;
    position_pair token{0, 0};
    while (tokenizer.next_token() && (token_idx < token_count)) {
      token = tokenizer.get_token();
      d_tokens[d_strings.size() * (token_idx++)] =
        string_index_pair{d_str.data() + token.first, (token.second - token.first)};
    }
    if (token_count == max_tokens)
      d_tokens[d_strings.size() * (token_idx - 1)] =
        string_index_pair{d_str.data() + token.first, (d_str.size_bytes() - token.first)};
  }

  whitespace_split_tokenizer_fn(column_device_view const& d_strings, size_type max_tokens)
    : base_whitespace_split_tokenizer(d_strings, max_tokens)
  {
  }
};

/**
 * @brief The tokenizer functions for rsplit() with whitespace.
 *
 * The whitespace tokenizer has no delimiter and handles one or more
 * consecutive whitespace characters as a single delimiter.
 *
 * This one processes tokens from the end of each string.
 */
struct whitespace_rsplit_tokenizer_fn : base_whitespace_split_tokenizer {
  /**
   * @brief This will create tokens around each runs of whitespace characters.
   *
   * Each token is placed in `d_all_tokens` so they align consecutively
   * with other tokens for the same output column.
   * That is, `d_tokens[col * strings_count + string_index]` is the token at column `col`
   * for string at `string_index`.
   *
   * @param idx Index of the string to process
   * @param column_count Number of output columns
   * @param d_token_counts Token counts for each string
   * @param d_all_tokens All output tokens for the strings column
   */
  __device__ void process_tokens(size_type idx,  // string position index
                                 size_type column_count,
                                 size_type const* d_token_counts,
                                 string_index_pair* d_all_tokens) const
  {
    string_index_pair* d_tokens = d_all_tokens + idx;
    if (d_strings.is_null(idx)) return;
    string_view const d_str = d_strings.element<cudf::string_view>(idx);
    if (d_str.empty()) return;
    whitespace_string_tokenizer tokenizer(d_str, true);
    size_type token_count = d_token_counts[idx];
    size_type token_idx   = 0;
    position_pair token{0, 0};
    while (tokenizer.prev_token() && (token_idx < token_count)) {
      token = tokenizer.get_token();
      d_tokens[d_strings.size() * (token_count - 1 - token_idx)] =
        string_index_pair{d_str.data() + token.first, (token.second - token.first)};
      ++token_idx;
    }
    if (token_count == max_tokens)
      d_tokens[d_strings.size() * (token_count - token_idx)] =
        string_index_pair{d_str.data(), token.second};
  }

  whitespace_rsplit_tokenizer_fn(column_device_view const& d_strings, size_type max_tokens)
    : base_whitespace_split_tokenizer(d_strings, max_tokens)
  {
  }
};

/**
 * @brief Generic split function called by split() and rsplit() using whitespace as a delimiter.
 *
 * The number of tokens for each string is computed by counting consecutive characters
 * between runs of whitespace in each string. The number of output columns is determined
 * by the string with the most tokens. Next the string_index_pairs for the entire column
 * is created.
 *
 * Finally, each column is built by creating a vector of tokens (string_index_pairs)
 * according to their position in each string. The first token from each string goes
 * into the first output column, the 2nd token from each string goes into the 2nd
 * output column, etc.
 *
 * This can be compared to Pandas `split()` with no delimiter and with `expand=True` but
 * with the rows/columns transposed.
 *
 *  import pandas as pd
 *  pd_series = pd.Series(['', None, 'a b', ' a b ', '  aa  bb  ', ' a  bbb   c', ' aa b  ccc  '])
 *  print(pd_series.str.split(pat=None, expand=True))
 *            0     1     2
 *      0  None  None  None
 *      1  None  None  None
 *      2     a     b  None
 *      3     a     b  None
 *      4    aa    bb  None
 *      5     a   bbb     c
 *      6    aa     b   ccc
 *
 *  print(pd_series.str.split(pat=None, n=1, expand=True))
 *            0         1
 *      0  None      None
 *      1  None      None
 *      2     a         b
 *      3     a        b
 *      4    aa      bb
 *      5     a   bbb   c
 *      6    aa  b  ccc
 *
 *  print(pd_series.str.split(pat=None, n=2, expand=True))
 *            0     1      2
 *      0  None  None   None
 *      1  None  None   None
 *      2     a     b   None
 *      3     a     b   None
 *      4    aa    bb   None
 *      5     a   bbb      c
 *      6    aa     b  ccc
 *
 * @tparam Tokenizer provides unique functions for split/rsplit.
 * @param strings_count The number of strings in the column
 * @param tokenizer Tokenizer for counting and producing tokens
 * @return table of columns for the output of the split
 */
template <typename Tokenizer>
std::unique_ptr<table> whitespace_split_fn(size_type strings_count,
                                           Tokenizer tokenizer,
                                           rmm::cuda_stream_view stream,
                                           rmm::mr::device_memory_resource* mr)
{
  auto execpol = rmm::exec_policy(stream);

  // compute the number of tokens per string
  size_type columns_count = 0;
  rmm::device_vector<size_type> token_counts(strings_count);
  auto d_token_counts = token_counts.data().get();
  if (strings_count > 0) {
    thrust::transform(
      execpol->on(stream.value()),
      thrust::make_counting_iterator<size_type>(0),
      thrust::make_counting_iterator<size_type>(strings_count),
      d_token_counts,
      [tokenizer] __device__(size_type idx) { return tokenizer.count_tokens(idx); });
    // column count is the maximum number of tokens for any string
    columns_count =
      *thrust::max_element(execpol->on(stream.value()), token_counts.begin(), token_counts.end());
  }

  std::vector<std::unique_ptr<column>> results;
  // boundary case: if no columns, return one null column (issue #119)
  if (columns_count == 0) {
    results.push_back(std::make_unique<column>(
      data_type{type_id::STRING},
      strings_count,
      rmm::device_buffer{0, stream, mr},  // no data
      cudf::detail::create_null_mask(strings_count, mask_state::ALL_NULL, stream, mr),
      strings_count));
  }

  // get the positions for every token
  rmm::device_vector<string_index_pair> tokens(columns_count * strings_count);
  string_index_pair* d_tokens = tokens.data().get();
  thrust::fill(execpol->on(stream.value()),
               d_tokens,
               d_tokens + (columns_count * strings_count),
               string_index_pair{nullptr, 0});
  thrust::for_each_n(
    execpol->on(stream.value()),
    thrust::make_counting_iterator<size_type>(0),
    strings_count,
    [tokenizer, columns_count, d_token_counts, d_tokens] __device__(size_type idx) {
      tokenizer.process_tokens(idx, columns_count, d_token_counts, d_tokens);
    });

  // Create each column.
  // - Each pair points to a string for that column for each row.
  // - Create the strings column from the vector using the strings factory.
  for (size_type col = 0; col < columns_count; ++col) {
    auto column_tokens = d_tokens + (col * strings_count);
    results.emplace_back(
      make_strings_column(column_tokens, column_tokens + strings_count, stream, mr));
  }
  return std::make_unique<table>(std::move(results));
}

}  // namespace

std::unique_ptr<table> split(
  strings_column_view const& strings_column,
  string_scalar const& delimiter      = string_scalar(""),
  size_type maxsplit                  = -1,
  rmm::cuda_stream_view stream        = rmm::cuda_stream_default,
  rmm::mr::device_memory_resource* mr = rmm::mr::get_current_device_resource())
{
  CUDF_EXPECTS(delimiter.is_valid(), "Parameter delimiter must be valid");

  size_type max_tokens = 0;
  if (maxsplit > 0) max_tokens = maxsplit + 1;  // makes consistent with Pandas

  auto strings_device_view = column_device_view::create(strings_column.parent(), stream);
  if (delimiter.size() == 0) {
    return whitespace_split_fn(strings_column.size(),
                               whitespace_split_tokenizer_fn{*strings_device_view, max_tokens},
                               stream,
                               mr);
  }

  string_view d_delimiter(delimiter.data(), delimiter.size());
  return split_fn(
    strings_column, split_tokenizer_fn{*strings_device_view, d_delimiter, max_tokens}, stream, mr);
}

std::unique_ptr<table> rsplit(
  strings_column_view const& strings_column,
  string_scalar const& delimiter      = string_scalar(""),
  size_type maxsplit                  = -1,
  rmm::cuda_stream_view stream        = rmm::cuda_stream_default,
  rmm::mr::device_memory_resource* mr = rmm::mr::get_current_device_resource())
{
  CUDF_EXPECTS(delimiter.is_valid(), "Parameter delimiter must be valid");

  size_type max_tokens = 0;
  if (maxsplit > 0) max_tokens = maxsplit + 1;  // makes consistent with Pandas

  auto strings_device_view = column_device_view::create(strings_column.parent(), stream);
  if (delimiter.size() == 0) {
    return whitespace_split_fn(strings_column.size(),
                               whitespace_rsplit_tokenizer_fn{*strings_device_view, max_tokens},
                               stream,
                               mr);
  }

  string_view d_delimiter(delimiter.data(), delimiter.size());
  return split_fn(
    strings_column, rsplit_tokenizer_fn{*strings_device_view, d_delimiter, max_tokens}, stream, mr);
}

}  // namespace detail

// external APIs

std::unique_ptr<table> split(strings_column_view const& strings_column,
                             string_scalar const& delimiter,
                             size_type maxsplit,
                             rmm::mr::device_memory_resource* mr)
{
  CUDF_FUNC_RANGE();
  return detail::split(strings_column, delimiter, maxsplit, rmm::cuda_stream_default, mr);
}

std::unique_ptr<table> rsplit(strings_column_view const& strings_column,
                              string_scalar const& delimiter,
                              size_type maxsplit,
                              rmm::mr::device_memory_resource* mr)
{
  CUDF_FUNC_RANGE();
  return detail::rsplit(strings_column, delimiter, maxsplit, rmm::cuda_stream_default, mr);
}

}  // namespace strings
}  // namespace cudf
