#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2020-2023, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "split.cuh"

#include <cudf/column/column.hpp>
#include <cudf/column/column_device_view.cuh>
#include <cudf/column/column_factories.hpp>
#include <cudf/detail/get_value.cuh>
#include <cudf/detail/nvtx/ranges.hpp>
#include <cudf/lists/detail/lists_column_factories.hpp>
#include <cudf/strings/detail/split_utils.cuh>
#include <cudf/strings/detail/strings_column_factories.cuh>
#include <cudf/strings/split/split.hpp>
#include <cudf/strings/string_view.cuh>
#include <cudf/utilities/default_stream.hpp>

#include <rmm/cuda_stream_view.hpp>

#include <thrust/for_each.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/scan.h>
#include <thrust/transform.h>

namespace cudf {
namespace strings {
namespace detail {

namespace {

template <typename Tokenizer>
std::unique_ptr<column> split_record_fn(strings_column_view const& input,
                                        Tokenizer tokenizer,
                                        rmm::cuda_stream_view stream,
                                        rmm::mr::device_memory_resource* mr)
{
  if (input.is_empty()) {
    return cudf::lists::detail::make_empty_lists_column(data_type{type_id::STRING}, stream, mr);
  }
  if (input.size() == input.null_count()) {
    auto offsets = std::make_unique<column>(input.offsets(), stream, mr);
    auto results = make_empty_column(type_id::STRING);
    return make_lists_column(input.size(),
                             std::move(offsets),
                             std::move(results),
                             input.null_count(),
                             copy_bitmask(input.parent(), stream, mr),
                             stream,
                             mr);
  }

  // builds the offsets and the vector of all tokens
  auto [offsets, tokens] = split_helper(input, tokenizer, stream, mr);

  // build a strings column from the tokens
  auto strings_child = make_strings_column(tokens.begin(), tokens.end(), stream, mr);

  return make_lists_column(input.size(),
                           std::move(offsets),
                           std::move(strings_child),
                           input.null_count(),
                           copy_bitmask(input.parent(), stream, mr),
                           stream,
                           mr);
}

enum class Direction { FORWARD, BACKWARD };

/**
 * @brief Identify the tokens from the `idx'th` string element of `d_strings`.
 */
template <Direction direction>
struct whitespace_token_reader_fn {
  column_device_view const d_strings;  // strings to split
  size_type const max_tokens = std::numeric_limits<size_type>::max();
  size_type const* d_token_offsets{};
  string_index_pair* d_tokens{};

  __device__ size_type count_tokens(size_type idx) const
  {
    if (d_strings.is_null(idx)) { return 0; }
    auto const d_str = d_strings.element<string_view>(idx);
    return count_tokens_whitespace(d_str, max_tokens);
  }

  __device__ void operator()(size_type idx)
  {
    auto const token_offset = d_token_offsets[idx];
    auto const token_count  = d_token_offsets[idx + 1] - token_offset;
    if (token_count == 0) { return; }
    auto d_result = d_tokens + token_offset;

    auto const d_str = d_strings.element<string_view>(idx);
    whitespace_string_tokenizer tokenizer(d_str, direction != Direction::FORWARD);
    size_type token_idx = 0;
    position_pair token{0, 0};
    if constexpr (direction == Direction::FORWARD) {
      while (tokenizer.next_token() && (token_idx < token_count)) {
        token = tokenizer.get_token();
        d_result[token_idx++] =
          string_index_pair{d_str.data() + token.first, token.second - token.first};
      }
      --token_idx;
      token.second = d_str.size_bytes() - token.first;
    } else {
      while (tokenizer.prev_token() && (token_idx < token_count)) {
        token = tokenizer.get_token();
        d_result[token_count - 1 - token_idx] =
          string_index_pair{d_str.data() + token.first, token.second - token.first};
        ++token_idx;
      }
      token_idx   = token_count - token_idx;  // token_count - 1 - (token_idx-1)
      token.first = 0;
    }
    // reset last token only if we hit the max
    if (token_count == max_tokens)
      d_result[token_idx] = string_index_pair{d_str.data() + token.first, token.second};
  }
};

}  // namespace

// The output is one list item per string
template <typename TokenReader>
std::unique_ptr<column> whitespace_split_record_fn(strings_column_view const& input,
                                                   TokenReader reader,
                                                   rmm::cuda_stream_view stream,
                                                   rmm::mr::device_memory_resource* mr)
{
  // create offsets column by counting the number of tokens per string
  auto sizes_itr = cudf::detail::make_counting_transform_iterator(
    0, [reader] __device__(auto idx) { return reader.count_tokens(idx); });
  auto [offsets, total_tokens] =
    cudf::detail::make_offsets_child_column(sizes_itr, sizes_itr + input.size(), stream, mr);
  auto d_offsets = offsets->view().template data<cudf::size_type>();

  // split each string into an array of index-pair values
  rmm::device_uvector<string_index_pair> tokens(total_tokens, stream);
  reader.d_token_offsets = d_offsets;
  reader.d_tokens        = tokens.data();
  thrust::for_each_n(
    rmm::exec_policy(stream), thrust::make_counting_iterator<size_type>(0), input.size(), reader);
  // convert the index-pairs into one big strings column
  auto strings_output = make_strings_column(tokens.begin(), tokens.end(), stream, mr);
  // create a lists column using the offsets and the strings columns
  return make_lists_column(input.size(),
                           std::move(offsets),
                           std::move(strings_output),
                           input.null_count(),
                           copy_bitmask(input.parent(), stream, mr),
                           stream,
                           mr);
}

template <Direction direction>
std::unique_ptr<column> split_record(strings_column_view const& strings,
                                     string_scalar const& delimiter,
                                     size_type maxsplit,
                                     rmm::cuda_stream_view stream,
                                     rmm::mr::device_memory_resource* mr)
{
  CUDF_EXPECTS(delimiter.is_valid(stream), "Parameter delimiter must be valid");

  // makes consistent with Pandas
  size_type max_tokens = maxsplit > 0 ? maxsplit + 1 : std::numeric_limits<size_type>::max();

  auto d_strings_column_ptr = column_device_view::create(strings.parent(), stream);
  if (delimiter.size() == 0) {
    return whitespace_split_record_fn(
      strings,
      whitespace_token_reader_fn<direction>{*d_strings_column_ptr, max_tokens},
      stream,
      mr);
  } else {
    string_view d_delimiter(delimiter.data(), delimiter.size());
    if (direction == Direction::FORWARD) {
      return split_record_fn(
        strings, split_tokenizer_fn{*d_strings_column_ptr, d_delimiter, max_tokens}, stream, mr);
    } else {
      return split_record_fn(
        strings, rsplit_tokenizer_fn{*d_strings_column_ptr, d_delimiter, max_tokens}, stream, mr);
    }
  }
}

}  // namespace detail

// external APIs

std::unique_ptr<column> split_record(strings_column_view const& strings,
                                     string_scalar const& delimiter,
                                     size_type maxsplit,
                                     rmm::cuda_stream_view stream,
                                     rmm::mr::device_memory_resource* mr)
{
  CUDF_FUNC_RANGE();
  return detail::split_record<detail::Direction::FORWARD>(strings, delimiter, maxsplit, stream, mr);
}

std::unique_ptr<column> rsplit_record(strings_column_view const& strings,
                                      string_scalar const& delimiter,
                                      size_type maxsplit,
                                      rmm::cuda_stream_view stream,
                                      rmm::mr::device_memory_resource* mr)
{
  CUDF_FUNC_RANGE();
  return detail::split_record<detail::Direction::BACKWARD>(
    strings, delimiter, maxsplit, stream, mr);
}

}  // namespace strings
}  // namespace cudf
