#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2020, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <strings/split/split_utils.cuh>

#include <cudf/column/column.hpp>
#include <cudf/column/column_device_view.cuh>
#include <cudf/column/column_factories.hpp>
#include <cudf/detail/get_value.cuh>
#include <cudf/detail/nvtx/ranges.hpp>
#include <cudf/strings/detail/strings_column_factories.cuh>
#include <cudf/strings/split/split.hpp>
#include <cudf/strings/string_view.cuh>
#include <cudf/strings/strings_column_view.hpp>

#include <rmm/cuda_stream_view.hpp>

#include <thrust/scan.h>
#include <thrust/transform.h>

namespace cudf {
namespace strings {
namespace detail {

using string_index_pair = thrust::pair<const char*, size_type>;

namespace {

enum class Dir { FORWARD, BACKWARD };

/**
 * @brief Compute the number of tokens for the `idx'th` string element of `d_strings`.
 *
 * The number of tokens is the same regardless if counting from the beginning
 * or the end of the string.
 */
struct token_counter_fn {
  column_device_view const d_strings;  // strings to split
  string_view const d_delimiter;       // delimiter for split
  size_type const max_tokens = std::numeric_limits<size_type>::max();

  __device__ size_type operator()(size_type idx) const
  {
    if (d_strings.is_null(idx)) { return 0; }

    auto const d_str      = d_strings.element<string_view>(idx);
    size_type token_count = 0;
    size_type start_pos   = 0;
    while (token_count < max_tokens - 1) {
      auto const delimiter_pos = d_str.find(d_delimiter, start_pos);
      if (delimiter_pos < 0) break;
      token_count++;
      start_pos = delimiter_pos + d_delimiter.length();
    }
    return token_count + 1;  // always at least one token
  }
};

/**
 * @brief Identify the tokens from the `idx'th` string element of `d_strings`.
 */
template <Dir dir>
struct token_reader_fn {
  column_device_view const d_strings;  // strings to split
  string_view const d_delimiter;       // delimiter for split
  int32_t* d_token_offsets{};          // for locating tokens in d_tokens
  string_index_pair* d_tokens{};

  __device__ string_index_pair resolve_token(string_view const& d_str,
                                             size_type start_pos,
                                             size_type end_pos,
                                             size_type delimiter_pos) const
  {
    if (dir == Dir::FORWARD) {
      auto const byte_offset = d_str.byte_offset(start_pos);
      return string_index_pair{d_str.data() + byte_offset,
                               d_str.byte_offset(delimiter_pos) - byte_offset};
    } else {
      auto const byte_offset = d_str.byte_offset(delimiter_pos + d_delimiter.length());
      return string_index_pair{d_str.data() + byte_offset,
                               d_str.byte_offset(end_pos) - byte_offset};
    }
  }

  __device__ void operator()(size_type idx)
  {
    if (d_strings.is_null(idx)) { return; }

    auto const token_offset = d_token_offsets[idx];
    auto const token_count  = d_token_offsets[idx + 1] - token_offset;
    auto d_result           = d_tokens + token_offset;
    auto const d_str        = d_strings.element<string_view>(idx);
    if (d_str.empty()) {
      // Pandas str.split("") for non-whitespace delimiter is an empty string
      *d_result = string_index_pair{"", 0};
      return;
    }

    size_type token_idx = 0;
    size_type start_pos = 0;               // updates only if moving forward
    size_type end_pos   = d_str.length();  // updates only if moving backward
    while (token_idx < token_count - 1) {
      auto const delimiter_pos = dir == Dir::FORWARD ? d_str.find(d_delimiter, start_pos)
                                                     : d_str.rfind(d_delimiter, start_pos, end_pos);
      if (delimiter_pos < 0) break;
      auto const token = resolve_token(d_str, start_pos, end_pos, delimiter_pos);
      if (dir == Dir::FORWARD) {
        d_result[token_idx] = token;
        start_pos           = delimiter_pos + d_delimiter.length();
      } else {
        d_result[token_count - 1 - token_idx] = token;
        end_pos                               = delimiter_pos;
      }
      token_idx++;
    }

    // set last token to remainder of the string
    if (dir == Dir::FORWARD) {
      auto const offset_bytes = d_str.byte_offset(start_pos);
      d_result[token_idx] =
        string_index_pair{d_str.data() + offset_bytes, d_str.byte_offset(end_pos) - offset_bytes};
    } else {
      d_result[0] = string_index_pair{d_str.data(), d_str.byte_offset(end_pos)};
    }
  }
};

/**
 * @brief Compute the number of tokens for the `idx'th` string element of `d_strings`.
 */
struct whitespace_token_counter_fn {
  column_device_view const d_strings;  // strings to split
  size_type const max_tokens = std::numeric_limits<size_type>::max();

  __device__ size_type operator()(size_type idx) const
  {
    if (d_strings.is_null(idx)) { return 0; }

    auto const d_str        = d_strings.element<string_view>(idx);
    size_type token_count   = 0;
    auto spaces             = true;
    auto reached_max_tokens = false;
    for (auto ch : d_str) {
      if (spaces != (ch <= ' ')) {
        if (!spaces) {
          if (token_count < max_tokens - 1) {
            token_count++;
          } else {
            reached_max_tokens = true;
            break;
          }
        }
        spaces = !spaces;
      }
    }
    // pandas.Series.str.split("") returns 0 tokens.
    if (reached_max_tokens || !spaces) token_count++;
    return token_count;
  }
};

/**
 * @brief Identify the tokens from the `idx'th` string element of `d_strings`.
 */
template <Dir dir>
struct whitespace_token_reader_fn {
  column_device_view const d_strings;  // strings to split
  size_type const max_tokens{};
  int32_t* d_token_offsets{};
  string_index_pair* d_tokens{};

  __device__ void operator()(size_type idx)
  {
    auto const token_offset = d_token_offsets[idx];
    auto const token_count  = d_token_offsets[idx + 1] - token_offset;
    if (token_count == 0) { return; }
    auto d_result = d_tokens + token_offset;

    auto const d_str = d_strings.element<string_view>(idx);
    whitespace_string_tokenizer tokenizer(d_str, dir != Dir::FORWARD);
    size_type token_idx = 0;
    position_pair token{0, 0};
    if (dir == Dir::FORWARD) {
      while (tokenizer.next_token() && (token_idx < token_count)) {
        token = tokenizer.get_token();
        d_result[token_idx++] =
          string_index_pair{d_str.data() + token.first, token.second - token.first};
      }
      --token_idx;
      token.second = d_str.size_bytes() - token.first;
    } else {
      while (tokenizer.prev_token() && (token_idx < token_count)) {
        token = tokenizer.get_token();
        d_result[token_count - 1 - token_idx] =
          string_index_pair{d_str.data() + token.first, token.second - token.first};
        ++token_idx;
      }
      token_idx   = token_count - token_idx;  // token_count - 1 - (token_idx-1)
      token.first = 0;
    }
    // reset last token only if we hit the max
    if (token_count == max_tokens)
      d_result[token_idx] = string_index_pair{d_str.data() + token.first, token.second};
  }
};

}  // namespace

// The output is one list item per string
template <typename TokenCounter, typename TokenReader>
std::unique_ptr<column> split_record_fn(strings_column_view const& strings,
                                        TokenCounter counter,
                                        TokenReader reader,
                                        rmm::cuda_stream_view stream,
                                        rmm::mr::device_memory_resource* mr)
{
  // create offsets column by counting the number of tokens per string
  auto strings_count = strings.size();
  auto offsets       = make_numeric_column(
    data_type{type_id::INT32}, strings_count + 1, mask_state::UNALLOCATED, stream, mr);
  auto d_offsets = offsets->mutable_view().data<int32_t>();
  thrust::transform(rmm::exec_policy(stream)->on(stream.value()),
                    thrust::make_counting_iterator(0),
                    thrust::make_counting_iterator(strings_count),
                    d_offsets,
                    counter);
  thrust::exclusive_scan(rmm::exec_policy(stream)->on(stream.value()),
                         d_offsets,
                         d_offsets + strings_count + 1,
                         d_offsets);

  // last entry is the total number of tokens to be generated
  auto total_tokens = cudf::detail::get_value<int32_t>(offsets->view(), strings_count, stream);
  // split each string into an array of index-pair values
  rmm::device_vector<string_index_pair> tokens(total_tokens);
  reader.d_token_offsets = d_offsets;
  reader.d_tokens        = tokens.data().get();
  thrust::for_each_n(rmm::exec_policy(stream)->on(stream.value()),
                     thrust::make_counting_iterator<size_type>(0),
                     strings_count,
                     reader);
  // convert the index-pairs into one big strings column
  auto strings_output = make_strings_column(tokens.begin(), tokens.end(), stream, mr);
  // create a lists column using the offsets and the strings columns
  return make_lists_column(strings_count,
                           std::move(offsets),
                           std::move(strings_output),
                           strings.null_count(),
                           copy_bitmask(strings.parent(), stream, mr));
}

template <Dir dir>
std::unique_ptr<column> split_record(
  strings_column_view const& strings,
  string_scalar const& delimiter      = string_scalar(""),
  size_type maxsplit                  = -1,
  rmm::cuda_stream_view stream        = rmm::cuda_stream_default,
  rmm::mr::device_memory_resource* mr = rmm::mr::get_current_device_resource())
{
  CUDF_EXPECTS(delimiter.is_valid(), "Parameter delimiter must be valid");

  // makes consistent with Pandas
  size_type max_tokens = maxsplit > 0 ? maxsplit + 1 : std::numeric_limits<size_type>::max();

  auto d_strings_column_ptr = column_device_view::create(strings.parent(), stream);
  if (delimiter.size() == 0) {
    return split_record_fn(strings,
                           whitespace_token_counter_fn{*d_strings_column_ptr, max_tokens},
                           whitespace_token_reader_fn<dir>{*d_strings_column_ptr, max_tokens},
                           stream,
                           mr);
  } else {
    string_view d_delimiter(delimiter.data(), delimiter.size());
    return split_record_fn(strings,
                           token_counter_fn{*d_strings_column_ptr, d_delimiter, max_tokens},
                           token_reader_fn<dir>{*d_strings_column_ptr, d_delimiter},
                           stream,
                           mr);
  }
}

}  // namespace detail

// external APIs

std::unique_ptr<column> split_record(strings_column_view const& strings,
                                     string_scalar const& delimiter,
                                     size_type maxsplit,
                                     rmm::mr::device_memory_resource* mr)
{
  CUDF_FUNC_RANGE();
  return detail::split_record<detail::Dir::FORWARD>(
    strings, delimiter, maxsplit, rmm::cuda_stream_default, mr);
}

std::unique_ptr<column> rsplit_record(strings_column_view const& strings,
                                      string_scalar const& delimiter,
                                      size_type maxsplit,
                                      rmm::mr::device_memory_resource* mr)
{
  CUDF_FUNC_RANGE();
  return detail::split_record<detail::Dir::BACKWARD>(
    strings, delimiter, maxsplit, rmm::cuda_stream_default, mr);
}

}  // namespace strings
}  // namespace cudf
