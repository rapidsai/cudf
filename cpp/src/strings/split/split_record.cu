#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2020-2023, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <cudf/column/column.hpp>
#include <cudf/column/column_device_view.cuh>
#include <cudf/column/column_factories.hpp>
#include <cudf/detail/get_value.cuh>
#include <cudf/detail/nvtx/ranges.hpp>
#include <cudf/detail/sizes_to_offsets_iterator.cuh>
#include <cudf/detail/utilities/device_atomics.cuh>
#include <cudf/strings/detail/split_utils.cuh>
#include <cudf/strings/detail/strings_column_factories.cuh>
#include <cudf/strings/split/split.hpp>
#include <cudf/strings/string_view.cuh>
#include <cudf/strings/strings_column_view.hpp>
#include <cudf/utilities/default_stream.hpp>
#include <cudf/utilities/span.hpp>

#include <rmm/cuda_stream_view.hpp>

#include <thrust/for_each.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/scan.h>
#include <thrust/transform.h>

namespace cudf {
namespace strings {
namespace detail {

namespace {

/**
 * @brief Base class for delimiter-based tokenizers
 *
 * These are common methods used by both split and rsplit tokenizer functors.
 */
template <typename Derived>
struct base_split_tokenizer {
  __device__ const char* get_base_ptr() const
  {
    return d_strings.child(strings_column_view::chars_column_index).data<char>();
  }

  __device__ string_view const get_string(size_type idx) const
  {
    return d_strings.element<string_view>(idx);
  }

  __device__ bool is_valid(size_type idx) const { return d_strings.is_valid(idx); }

  /**
   * @brief This counts the tokens for strings that contain delimiters
   *
   * Counting tokens is the same regardless if counting from the left
   * or from the right. This logic counts from the left which is simpler
   * logic. The count will be truncated appropriately to the max_tokens value.
   *
   * @param idx Index of input string
   * @param d_positions Start positions of all the delimiters
   * @param d_delimiter_offsets Offsets per string to delimiters in d_positions
   */
  __device__ size_type count_tokens(size_type idx,
                                    size_type const* d_positions,
                                    size_type const* d_delimiter_offsets) const
  {
    if (!is_valid(idx)) { return 0; }

    auto const delim_size = d_delimiter.size_bytes();
    auto const d_str      = get_string(idx);
    auto const d_str_end  = d_str.data() + d_str.size_bytes();
    auto const base_ptr   = get_base_ptr() + delim_size - 1;
    auto const delimiters =
      cudf::device_span<size_type const>(d_positions + d_delimiter_offsets[idx],
                                         d_delimiter_offsets[idx + 1] - d_delimiter_offsets[idx]);

    size_type token_count = 1;  // all strings will have at least one token
    size_type last_pos    = delimiters[0] - delim_size;
    for (auto d_pos : delimiters) {
      // delimiter must fit in string && overlapping delimiters are ignored
      if (((base_ptr + d_pos) < d_str_end) && ((d_pos - last_pos) >= delim_size)) {
        ++token_count;
        last_pos = d_pos;
      }
    }
    // number of tokens is capped to max_tokens
    return ((max_tokens > 0) && (token_count > max_tokens)) ? max_tokens : token_count;
  }

  /**
   * @brief This will create tokens around each delimiter honoring the string boundaries
   * in which the delimiter resides
   *
   * Each token is placed in `d_all_tokens` so they align consecutively
   * with other tokens for the same output column.
   *
   * @param idx Index of the string to tokenize
   * @param d_tokens_offsets Token offsets for each string
   * @param d_positions The beginning byte position of each delimiter
   * @param d_delimiter_offsets Offsets to d_positions to each delimiter set per string
   * @param d_all_tokens All output tokens for the strings column
   */
  __device__ void get_tokens(size_type idx,
                             size_type const* d_tokens_offsets,
                             size_type const* d_positions,
                             size_type const* d_delimiter_offsets,
                             string_index_pair* d_all_tokens) const
  {
    auto const d_tokens =  // this string's tokens output
      cudf::device_span<string_index_pair>(d_all_tokens + d_tokens_offsets[idx],
                                           d_tokens_offsets[idx + 1] - d_tokens_offsets[idx]);

    if (!is_valid(idx)) {
      d_tokens[0] = string_index_pair{nullptr, 0};
      return;
    }

    auto const d_str = get_string(idx);

    // max_tokens already included in token counts
    if (d_tokens.size() == 1) {
      d_tokens[0] = string_index_pair{d_str.data(), d_str.size_bytes()};
      return;
    }

    auto const delimiters =
      cudf::device_span<size_type const>(d_positions + d_delimiter_offsets[idx],
                                         d_delimiter_offsets[idx + 1] - d_delimiter_offsets[idx]);

    auto& derived = static_cast<Derived const&>(*this);
    derived.process_tokens(d_str, delimiters, d_tokens);
  }

  base_split_tokenizer(column_device_view const& d_strings,
                       string_view const& d_delimiter,
                       size_type max_tokens)
    : d_strings(d_strings), d_delimiter(d_delimiter), max_tokens(max_tokens)
  {
  }

 protected:
  column_device_view const d_strings;  // strings to split
  string_view const d_delimiter;       // delimiter for split
  size_type max_tokens;
};

/**
 * @brief The tokenizer functions for split().
 *
 * The methods here count delimiters, tokens, and output token elements
 * for each string in a strings column.
 */
struct split_tokenizer_fn : base_split_tokenizer<split_tokenizer_fn> {
  /**
   * @brief Returns `true` if the byte at `idx` is the start of the delimiter
   *
   * @param idx Index of a byte in the chars column
   * @param d_offsets Offsets values to locate the chars ranges
   * @param chars_bytes Total number of characters to process
   * @return true if delimiter is found starting at position `idx`
   */
  __device__ bool is_delimiter(size_type idx,
                               size_type const* d_offsets,
                               size_type chars_bytes) const
  {
    auto const d_chars = get_base_ptr() + d_offsets[0];
    if (idx + d_delimiter.size_bytes() > chars_bytes) { return false; }
    return d_delimiter.compare(d_chars + idx, d_delimiter.size_bytes()) == 0;
  }

  /**
   * @brief This will create tokens around each delimiter honoring the string boundaries
   * in which the delimiter resides
   *
   * @param d_str String to tokenize
   * @param d_delimiters Positions of delimiters for this string
   * @param d_tokens Output vector to store tokens for this string
   */
  __device__ void process_tokens(string_view const d_str,
                                 device_span<size_type const> d_delimiters,
                                 device_span<string_index_pair> d_tokens) const
  {
    auto const base_ptr    = get_base_ptr();  // d_positions values based on this
    auto str_ptr           = d_str.data();
    auto const str_end     = str_ptr + d_str.size_bytes();  // end of the string
    auto const token_count = static_cast<size_type>(d_tokens.size());
    auto const delim_count = static_cast<size_type>(d_delimiters.size());

    // build the index-pair of each token for this string
    for (size_type t = 0; t < token_count; ++t) {
      auto next_delim = (t < delim_count)                 // bounds check for delims in last string
                          ? (base_ptr + d_delimiters[t])  // start of next delimiter
                          : str_end;                      // or end of this string

      auto eptr = (next_delim < str_end)        // make sure delimiter is inside this string
                      && (t + 1 < token_count)  // and this is not the last token
                    ? next_delim
                    : str_end;

      // store the token into the output vector
      d_tokens[t] =
        string_index_pair{str_ptr, static_cast<size_type>(thrust::distance(str_ptr, eptr))};

      // setup for next token
      str_ptr = eptr + d_delimiter.size_bytes();
    }
  }

  split_tokenizer_fn(column_device_view const& d_strings,
                     string_view const& d_delimiter,
                     size_type max_tokens)
    : base_split_tokenizer(d_strings, d_delimiter, max_tokens)
  {
  }
};

/**
 * @brief The tokenizer functions for split_record
 *
 * The methods here identify delimiters and output token elements
 * for each string in a strings column.
 *
 * Same as split_tokenizer_fn except delimiters are searched from the end of each string.
 */
struct rsplit_tokenizer_fn : base_split_tokenizer<rsplit_tokenizer_fn> {
  /**
   * @brief Returns `true` if the byte at `idx` is the end of the delimiter
   *
   * @param idx Index of a byte in the chars column
   * @param d_offsets Offsets values to locate the chars ranges
   * @return true if delimiter is found ending at position `idx`
   */
  __device__ bool is_delimiter(size_type idx, size_type const* d_offsets, size_type) const
  {
    auto const delim_length = d_delimiter.size_bytes();
    if (idx < delim_length - 1) { return false; }
    auto const d_chars = get_base_ptr() + d_offsets[0];
    return d_delimiter.compare(d_chars + idx - (delim_length - 1), delim_length) == 0;
  }

  /**
   * @brief This will create tokens around each delimiter honoring the string boundaries
   * in which the delimiter resides
   *
   * The tokens are processed from the end of each string so the `max_tokens`
   * and any overlapping delimiters are honored correctly.
   *
   * @param d_str String to tokenize
   * @param d_delimiters Positions of delimiters for this string
   * @param d_tokens Output vector to store tokens for this string
   */
  __device__ void process_tokens(string_view const d_str,
                                 device_span<size_type const> d_delimiters,
                                 device_span<string_index_pair> d_tokens) const
  {
    auto const base_ptr    = get_base_ptr();  // d_positions values are based on this ptr
    auto const str_begin   = d_str.data();    // beginning of the string
    auto const token_count = static_cast<size_type>(d_tokens.size());
    auto const delim_count = static_cast<size_type>(d_delimiters.size());

    // build the index-pair of each token for this string
    auto str_ptr = str_begin + d_str.size_bytes();
    for (size_type t = 0; t < token_count; ++t) {
      auto prev_delim =
        (t < delim_count)                                       // boundary check;
          ? (base_ptr + d_delimiters[delim_count - 1 - t] + 1)  // end of prev delimiter
          : str_begin;                                          // or the start of this string

      auto sptr = (prev_delim > str_begin)      // make sure delimiter is inside the string
                      && (t + 1 < token_count)  // and this is not the last token
                    ? prev_delim
                    : str_begin;

      // store the token into the output -- building the array backwards
      d_tokens[(token_count - 1 - t)] =
        string_index_pair{sptr, static_cast<size_type>(str_ptr - sptr)};

      // setup for next/prev token
      str_ptr = sptr - d_delimiter.size_bytes();
    }
  }

  rsplit_tokenizer_fn(column_device_view const& d_strings,
                      string_view const& d_delimiter,
                      size_type max_tokens)
    : base_split_tokenizer(d_strings, d_delimiter, max_tokens)
  {
  }
};

}  // namespace

template <typename Tokenizer>
std::unique_ptr<column> split_record_fn(strings_column_view const& input,
                                        Tokenizer tokenizer,
                                        rmm::cuda_stream_view stream,
                                        rmm::mr::device_memory_resource* mr)
{
  auto const strings_count = input.size();
  if (strings_count == 0) { return make_empty_column(type_id::LIST); }
  if (strings_count == input.null_count()) {
    auto offsets = std::make_unique<column>(input.offsets(), stream, mr);
    auto results = std::make_unique<column>(input.parent(), stream, mr);
    return make_lists_column(strings_count,
                             std::move(offsets),
                             std::move(results),
                             input.null_count(),
                             copy_bitmask(input.parent(), stream, mr),
                             stream,
                             mr);
  }

  auto const chars_bytes =
    cudf::detail::get_value<size_type>(input.offsets(), input.offset() + strings_count, stream) -
    cudf::detail::get_value<size_type>(input.offsets(), input.offset(), stream);

  auto d_offsets = input.offsets_begin();

  // count the number of delimiters in the entire column
  auto const delimiter_count =
    thrust::count_if(rmm::exec_policy(stream),
                     thrust::make_counting_iterator<size_type>(0),
                     thrust::make_counting_iterator<size_type>(chars_bytes),
                     [tokenizer, d_offsets, chars_bytes] __device__(size_type idx) {
                       return tokenizer.is_delimiter(idx, d_offsets, chars_bytes);
                     });

  // create vector of every delimiter position in the chars column
  auto delimiter_positions = rmm::device_uvector<size_type>(delimiter_count, stream);
  auto d_positions         = delimiter_positions.data();
  auto const copy_end =
    thrust::copy_if(rmm::exec_policy(stream),
                    thrust::make_counting_iterator<size_type>(0),
                    thrust::make_counting_iterator<size_type>(chars_bytes),
                    delimiter_positions.begin(),
                    [tokenizer, d_offsets, chars_bytes] __device__(size_type idx) {
                      return tokenizer.is_delimiter(idx, d_offsets, chars_bytes);
                    });

  // create a vector of offsets to each string's delimiter set within delimiter_positions
  auto const delimiter_offsets = [&] {
    // first, create a vector of string indices for each delimiter
    auto string_indices = rmm::device_uvector<size_type>(delimiter_count, stream);
    thrust::upper_bound(rmm::exec_policy(stream),
                        d_offsets,
                        d_offsets + strings_count,
                        delimiter_positions.begin(),
                        copy_end,
                        string_indices.begin());

    // compute delimiter offsets per string
    auto delimiter_offsets   = rmm::device_uvector<size_type>(strings_count + 1, stream);
    auto d_delimiter_offsets = delimiter_offsets.data();

    // memset required to zero-out any null-entries or strings with no delimiters
    hipMemsetAsync(d_delimiter_offsets, 0, delimiter_offsets.size() * sizeof(size_type), stream);

    // next, count the number of delimiters per string
    auto d_string_indices = string_indices.data();  // identifies strings with delimiters only
    thrust::for_each_n(rmm::exec_policy(stream),
                       thrust::make_counting_iterator<size_type>(0),
                       delimiter_count,
                       [d_string_indices, d_delimiter_offsets] __device__(size_type idx) {
                         auto const str_idx = d_string_indices[idx] - 1;
                         atomicAdd(d_delimiter_offsets + str_idx, 1);
                       });
    // finally, convert the counts into offsets
    thrust::exclusive_scan(rmm::exec_policy(stream),
                           delimiter_offsets.begin(),
                           delimiter_offsets.end(),
                           delimiter_offsets.begin());
    return delimiter_offsets;
  }();
  auto const d_delimiter_offsets = delimiter_offsets.data();

  auto token_counts = rmm::device_uvector<size_type>(strings_count, stream);

  // compute the number of tokens per string
  thrust::transform(
    rmm::exec_policy(stream),
    thrust::make_counting_iterator<size_type>(0),
    thrust::make_counting_iterator<size_type>(strings_count),
    token_counts.begin(),
    [tokenizer, d_positions, d_delimiter_offsets] __device__(size_type idx) -> size_type {
      return tokenizer.count_tokens(idx, d_positions, d_delimiter_offsets);
    });

  auto offsets = std::get<0>(
    cudf::detail::make_offsets_child_column(token_counts.begin(), token_counts.end(), stream, mr));
  auto const total_tokens =
    cudf::detail::get_value<size_type>(offsets->view(), strings_count, stream);
  auto const d_tokens_offsets = offsets->view().data<size_type>();

  // create working area to hold all token positions
  auto tokens   = rmm::device_uvector<string_index_pair>(total_tokens, stream);
  auto d_tokens = tokens.data();
  // fill in the token objects
  thrust::for_each_n(
    rmm::exec_policy(stream),
    thrust::make_counting_iterator<size_type>(0),
    strings_count,
    [tokenizer, d_tokens_offsets, d_positions, d_delimiter_offsets, d_tokens] __device__(
      size_type idx) {
      tokenizer.get_tokens(idx, d_tokens_offsets, d_positions, d_delimiter_offsets, d_tokens);
    });

  // build strings column from tokens
  auto strings_child = make_strings_column(tokens.begin(), tokens.end(), stream, mr);
  return make_lists_column(strings_count,
                           std::move(offsets),
                           std::move(strings_child),
                           input.null_count(),
                           copy_bitmask(input.parent(), stream, mr),
                           stream,
                           mr);

  // auto const columns_count = thrust::reduce(
  //  rmm::exec_policy(stream), token_counts.begin(), token_counts.end(), 0, thrust::maximum{});
  // std::cout << "count = " << columns_count << "\n";
  // size_type token_index = 10;
  // auto itr = cudf::detail::make_counting_transform_iterator(
  //  0, [d_tokens, d_tokens_offsets, token_index, columns_count] __device__(size_type idx) {
  //    auto offset      = d_tokens_offsets[idx];
  //    auto token_count = d_tokens_offsets[idx + 1] - offset;
  //    return (token_index < token_count) ? d_tokens[offset + token_index]
  //                                       : string_index_pair{nullptr, 0};
  //  });
  // return make_strings_column(itr, itr + strings_count, stream, mr);
}

namespace {

enum class Dir { FORWARD, BACKWARD };

/**
 * @brief Compute the number of tokens for the `idx'th` string element of `d_strings`.
 */
struct whitespace_token_counter_fn {
  column_device_view const d_strings;  // strings to split
  size_type const max_tokens = std::numeric_limits<size_type>::max();

  __device__ size_type operator()(size_type idx) const
  {
    if (d_strings.is_null(idx)) { return 0; }

    auto const d_str        = d_strings.element<string_view>(idx);
    size_type token_count   = 0;
    auto spaces             = true;
    auto reached_max_tokens = false;
    for (auto ch : d_str) {
      if (spaces != (ch <= ' ')) {
        if (!spaces) {
          if (token_count < max_tokens - 1) {
            token_count++;
          } else {
            reached_max_tokens = true;
            break;
          }
        }
        spaces = !spaces;
      }
    }
    // pandas.Series.str.split("") returns 0 tokens.
    if (reached_max_tokens || !spaces) token_count++;
    return token_count;
  }
};

/**
 * @brief Identify the tokens from the `idx'th` string element of `d_strings`.
 */
template <Dir dir>
struct whitespace_token_reader_fn {
  column_device_view const d_strings;  // strings to split
  size_type const max_tokens{};
  int32_t* d_token_offsets{};
  string_index_pair* d_tokens{};

  __device__ void operator()(size_type idx)
  {
    auto const token_offset = d_token_offsets[idx];
    auto const token_count  = d_token_offsets[idx + 1] - token_offset;
    if (token_count == 0) { return; }
    auto d_result = d_tokens + token_offset;

    auto const d_str = d_strings.element<string_view>(idx);
    whitespace_string_tokenizer tokenizer(d_str, dir != Dir::FORWARD);
    size_type token_idx = 0;
    position_pair token{0, 0};
    if constexpr (dir == Dir::FORWARD) {
      while (tokenizer.next_token() && (token_idx < token_count)) {
        token = tokenizer.get_token();
        d_result[token_idx++] =
          string_index_pair{d_str.data() + token.first, token.second - token.first};
      }
      --token_idx;
      token.second = d_str.size_bytes() - token.first;
    } else {
      while (tokenizer.prev_token() && (token_idx < token_count)) {
        token = tokenizer.get_token();
        d_result[token_count - 1 - token_idx] =
          string_index_pair{d_str.data() + token.first, token.second - token.first};
        ++token_idx;
      }
      token_idx   = token_count - token_idx;  // token_count - 1 - (token_idx-1)
      token.first = 0;
    }
    // reset last token only if we hit the max
    if (token_count == max_tokens)
      d_result[token_idx] = string_index_pair{d_str.data() + token.first, token.second};
  }
};

}  // namespace

// The output is one list item per string
template <typename TokenCounter, typename TokenReader>
std::unique_ptr<column> whitespace_split_record_fn(strings_column_view const& strings,
                                                   TokenCounter counter,
                                                   TokenReader reader,
                                                   rmm::cuda_stream_view stream,
                                                   rmm::mr::device_memory_resource* mr)
{
  // create offsets column by counting the number of tokens per string
  auto strings_count = strings.size();
  auto offsets       = make_numeric_column(
    data_type{type_id::INT32}, strings_count + 1, mask_state::UNALLOCATED, stream, mr);
  auto d_offsets = offsets->mutable_view().data<int32_t>();
  thrust::transform(rmm::exec_policy(stream),
                    thrust::make_counting_iterator<size_type>(0),
                    thrust::make_counting_iterator<size_type>(strings_count),
                    d_offsets,
                    counter);
  thrust::exclusive_scan(
    rmm::exec_policy(stream), d_offsets, d_offsets + strings_count + 1, d_offsets);

  // last entry is the total number of tokens to be generated
  auto total_tokens = cudf::detail::get_value<int32_t>(offsets->view(), strings_count, stream);
  // split each string into an array of index-pair values
  rmm::device_uvector<string_index_pair> tokens(total_tokens, stream);
  reader.d_token_offsets = d_offsets;
  reader.d_tokens        = tokens.data();
  thrust::for_each_n(
    rmm::exec_policy(stream), thrust::make_counting_iterator<size_type>(0), strings_count, reader);
  // convert the index-pairs into one big strings column
  auto strings_output = make_strings_column(tokens.begin(), tokens.end(), stream, mr);
  // create a lists column using the offsets and the strings columns
  return make_lists_column(strings_count,
                           std::move(offsets),
                           std::move(strings_output),
                           strings.null_count(),
                           copy_bitmask(strings.parent(), stream, mr),
                           stream,
                           mr);
}

template <Dir dir>
std::unique_ptr<column> split_record(strings_column_view const& strings,
                                     string_scalar const& delimiter,
                                     size_type maxsplit,
                                     rmm::cuda_stream_view stream,
                                     rmm::mr::device_memory_resource* mr)
{
  CUDF_EXPECTS(delimiter.is_valid(stream), "Parameter delimiter must be valid");

  // makes consistent with Pandas
  size_type max_tokens = maxsplit > 0 ? maxsplit + 1 : std::numeric_limits<size_type>::max();

  auto d_strings_column_ptr = column_device_view::create(strings.parent(), stream);
  if (delimiter.size() == 0) {
    return whitespace_split_record_fn(
      strings,
      whitespace_token_counter_fn{*d_strings_column_ptr, max_tokens},
      whitespace_token_reader_fn<dir>{*d_strings_column_ptr, max_tokens},
      stream,
      mr);
  } else {
    string_view d_delimiter(delimiter.data(), delimiter.size());
    if (dir == Dir::FORWARD) {
      return split_record_fn(
        strings, split_tokenizer_fn{*d_strings_column_ptr, d_delimiter, max_tokens}, stream, mr);
    } else {
      return split_record_fn(
        strings, rsplit_tokenizer_fn{*d_strings_column_ptr, d_delimiter, max_tokens}, stream, mr);
    }
  }
}

}  // namespace detail

// external APIs

std::unique_ptr<column> split_record(strings_column_view const& strings,
                                     string_scalar const& delimiter,
                                     size_type maxsplit,
                                     rmm::mr::device_memory_resource* mr)
{
  CUDF_FUNC_RANGE();
  return detail::split_record<detail::Dir::FORWARD>(
    strings, delimiter, maxsplit, cudf::get_default_stream(), mr);
}

std::unique_ptr<column> rsplit_record(strings_column_view const& strings,
                                      string_scalar const& delimiter,
                                      size_type maxsplit,
                                      rmm::mr::device_memory_resource* mr)
{
  CUDF_FUNC_RANGE();
  return detail::split_record<detail::Dir::BACKWARD>(
    strings, delimiter, maxsplit, cudf::get_default_stream(), mr);
}

}  // namespace strings
}  // namespace cudf
