#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2022, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <strings/count_matches.hpp>
#include <strings/regex/regex_program_impl.h>
#include <strings/regex/utilities.cuh>

#include <cudf/column/column.hpp>
#include <cudf/column/column_device_view.cuh>
#include <cudf/column/column_factories.hpp>
#include <cudf/detail/get_value.cuh>
#include <cudf/detail/iterator.cuh>
#include <cudf/detail/nvtx/ranges.hpp>
#include <cudf/strings/detail/strings_column_factories.cuh>
#include <cudf/strings/split/split_re.hpp>
#include <cudf/strings/string_view.cuh>
#include <cudf/utilities/default_stream.hpp>

#include <rmm/cuda_stream_view.hpp>

#include <thrust/distance.h>
#include <thrust/functional.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/pair.h>
#include <thrust/transform_reduce.h>
#include <thrust/transform_scan.h>

namespace cudf {
namespace strings {
namespace detail {
namespace {

using string_index_pair = thrust::pair<const char*, size_type>;

enum class split_direction {
  FORWARD,  ///< for split logic
  BACKWARD  ///< for rsplit logic
};

/**
 * @brief Identify the tokens from the `idx'th` string element of `d_strings`.
 *
 * Each string's tokens are stored in the `d_tokens` vector.
 * The `d_token_offsets` specifies the output position within `d_tokens`
 * for each string.
 */
struct token_reader_fn {
  column_device_view const d_strings;
  split_direction const direction;
  offset_type const* d_token_offsets;
  string_index_pair* d_tokens;

  __device__ void operator()(size_type const idx, reprog_device const prog, int32_t const prog_idx)
  {
    if (d_strings.is_null(idx)) { return; }
    auto const d_str = d_strings.element<string_view>(idx);

    auto const token_offset = d_token_offsets[idx];
    auto const token_count  = d_token_offsets[idx + 1] - token_offset;
    auto const d_result     = d_tokens + token_offset;  // store tokens here

    size_type token_idx = 0;
    size_type begin     = 0;  // characters
    size_type end       = -1;
    size_type last_pos  = 0;  // bytes
    while (prog.find(prog_idx, d_str, begin, end) > 0) {
      // get the token (characters just before this match)
      auto const token =
        string_index_pair{d_str.data() + last_pos, d_str.byte_offset(begin) - last_pos};
      // store it if we have space
      if (token_idx < token_count - 1) {
        d_result[token_idx++] = token;
      } else {
        if (direction == split_direction::FORWARD) { break; }  // we are done
        for (auto l = 0; l < token_idx - 1; ++l) {
          d_result[l] = d_result[l + 1];  // shift left
        }
        d_result[token_idx - 1] = token;
      }
      // setup for next match
      last_pos = d_str.byte_offset(end);
      begin    = end + (begin == end);
      end      = -1;
    }

    // set the last token to the remainder of the string
    d_result[token_idx] = string_index_pair{d_str.data() + last_pos, d_str.size_bytes() - last_pos};

    if (direction == split_direction::BACKWARD) {
      // update first entry -- this happens when max_tokens is hit before the end of the string
      auto const first_offset =
        d_result[0].first
          ? static_cast<size_type>(thrust::distance(d_str.data(), d_result[0].first))
          : 0;
      if (first_offset) {
        d_result[0] = string_index_pair{d_str.data(), first_offset + d_result[0].second};
      }
    }
  }
};

/**
 * @brief Call regex to split each input string into tokens.
 *
 * This will also convert the `offsets` values from counts to offsets.
 *
 * @param d_strings Strings to split
 * @param d_prog Regex to evaluate against each string
 * @param direction Whether tokens are generated forwards or backwards.
 * @param max_tokens The maximum number of tokens for each split.
 * @param offsets The number of matches on input.
 *                The offsets for each token in each string on output.
 * @param stream CUDA stream used for kernel launches.
 */
rmm::device_uvector<string_index_pair> generate_tokens(column_device_view const& d_strings,
                                                       reprog_device& d_prog,
                                                       split_direction direction,
                                                       size_type maxsplit,
                                                       mutable_column_view& offsets,
                                                       rmm::cuda_stream_view stream)
{
  auto const strings_count = d_strings.size();

  auto const max_tokens = maxsplit > 0 ? maxsplit : std::numeric_limits<size_type>::max();

  auto const begin     = thrust::make_counting_iterator<size_type>(0);
  auto const end       = thrust::make_counting_iterator<size_type>(strings_count);
  auto const d_offsets = offsets.data<offset_type>();

  // convert match counts to token offsets
  auto map_fn = [d_strings, d_offsets, max_tokens] __device__(auto idx) {
    return d_strings.is_null(idx) ? 0 : std::min(d_offsets[idx], max_tokens) + 1;
  };
  thrust::transform_exclusive_scan(
    rmm::exec_policy(stream), begin, end + 1, d_offsets, map_fn, 0, thrust::plus<offset_type>{});

  // the last offset entry is the total number of tokens to be generated
  auto const total_tokens = cudf::detail::get_value<offset_type>(offsets, strings_count, stream);

  rmm::device_uvector<string_index_pair> tokens(total_tokens, stream);
  if (total_tokens == 0) { return tokens; }

  launch_for_each_kernel(token_reader_fn{d_strings, direction, d_offsets, tokens.data()},
                         d_prog,
                         d_strings.size(),
                         stream);

  return tokens;
}

/**
 * @brief Returns string pair for the specified column for each string in `d_strings`
 *
 * This is used to build the table result of a split.
 * Null is returned if the row is null or if the `column_index` is larger
 * than the token count for that string.
 */
struct tokens_transform_fn {
  column_device_view const d_strings;
  string_index_pair const* d_tokens;
  offset_type const* d_token_offsets;
  size_type const column_index;

  __device__ string_index_pair operator()(size_type idx) const
  {
    auto const offset      = d_token_offsets[idx];
    auto const token_count = d_token_offsets[idx + 1] - offset;
    return (column_index >= token_count) || d_strings.is_null(idx)
             ? string_index_pair{nullptr, 0}
             : d_tokens[offset + column_index];
  }
};

std::unique_ptr<table> split_re(strings_column_view const& input,
                                regex_program const& prog,
                                split_direction direction,
                                size_type maxsplit,
                                rmm::cuda_stream_view stream,
                                rmm::mr::device_memory_resource* mr)
{
  CUDF_EXPECTS(!prog.pattern().empty(), "Parameter pattern must not be empty");

  auto const strings_count = input.size();

  std::vector<std::unique_ptr<column>> results;
  if (strings_count == 0) {
    results.push_back(make_empty_column(type_id::STRING));
    return std::make_unique<table>(std::move(results));
  }

  // create device object from regex_program
  auto d_prog = regex_device_builder::create_prog_device(prog, stream);

  auto d_strings = column_device_view::create(input.parent(), stream);

  // count the number of delimiters matched in each string
  auto offsets = count_matches(
    *d_strings, *d_prog, strings_count + 1, stream, rmm::mr::get_current_device_resource());
  auto offsets_view = offsets->mutable_view();
  auto d_offsets    = offsets_view.data<offset_type>();

  // get the split tokens from the input column; this also converts the counts into offsets
  auto tokens = generate_tokens(*d_strings, *d_prog, direction, maxsplit, offsets_view, stream);

  // the output column count is the maximum number of tokens generated for any input string
  auto const columns_count = thrust::transform_reduce(
    rmm::exec_policy(stream),
    thrust::make_counting_iterator<size_type>(0),
    thrust::make_counting_iterator<size_type>(strings_count),
    [d_offsets] __device__(auto const idx) -> size_type {
      return d_offsets[idx + 1] - d_offsets[idx];
    },
    0,
    thrust::maximum<size_type>{});

  // boundary case: if no columns, return one all-null column (custrings issue #119)
  if (columns_count == 0) {
    results.push_back(std::make_unique<column>(
      data_type{type_id::STRING},
      strings_count,
      rmm::device_buffer{0, stream, mr},  // no data
      cudf::detail::create_null_mask(strings_count, mask_state::ALL_NULL, stream, mr),
      strings_count));
    return std::make_unique<table>(std::move(results));
  }

  // convert the tokens into multiple strings columns
  auto make_strings_lambda = [&](size_type column_index) {
    // returns appropriate token for each row/column
    auto indices_itr = cudf::detail::make_counting_transform_iterator(
      0, tokens_transform_fn{*d_strings, tokens.data(), d_offsets, column_index});
    return make_strings_column(indices_itr, indices_itr + strings_count, stream, mr);
  };
  // build a vector of columns
  results.resize(columns_count);
  std::transform(thrust::make_counting_iterator<size_type>(0),
                 thrust::make_counting_iterator<size_type>(columns_count),
                 results.begin(),
                 make_strings_lambda);

  return std::make_unique<table>(std::move(results));
}

std::unique_ptr<column> split_record_re(strings_column_view const& input,
                                        regex_program const& prog,
                                        split_direction direction,
                                        size_type maxsplit,
                                        rmm::cuda_stream_view stream,
                                        rmm::mr::device_memory_resource* mr)
{
  CUDF_EXPECTS(!prog.pattern().empty(), "Parameter pattern must not be empty");

  auto const strings_count = input.size();

  // create device object from regex_program
  auto d_prog = regex_device_builder::create_prog_device(prog, stream);

  auto d_strings = column_device_view::create(input.parent(), stream);

  // count the number of delimiters matched in each string
  auto offsets      = count_matches(*d_strings, *d_prog, strings_count + 1, stream, mr);
  auto offsets_view = offsets->mutable_view();

  // get the split tokens from the input column; this also converts the counts into offsets
  auto tokens = generate_tokens(*d_strings, *d_prog, direction, maxsplit, offsets_view, stream);

  // convert the tokens into one big strings column
  auto strings_output = make_strings_column(tokens.begin(), tokens.end(), stream, mr);

  // create a lists column using the offsets and the strings columns
  return make_lists_column(strings_count,
                           std::move(offsets),
                           std::move(strings_output),
                           input.null_count(),
                           copy_bitmask(input.parent(), stream, mr),
                           stream,
                           mr);
}

}  // namespace

std::unique_ptr<table> split_re(strings_column_view const& input,
                                regex_program const& prog,
                                size_type maxsplit,
                                rmm::cuda_stream_view stream,
                                rmm::mr::device_memory_resource* mr)
{
  return split_re(input, prog, split_direction::FORWARD, maxsplit, stream, mr);
}

std::unique_ptr<column> split_record_re(strings_column_view const& input,
                                        regex_program const& prog,
                                        size_type maxsplit,
                                        rmm::cuda_stream_view stream,
                                        rmm::mr::device_memory_resource* mr)
{
  return split_record_re(input, prog, split_direction::FORWARD, maxsplit, stream, mr);
}

std::unique_ptr<table> rsplit_re(strings_column_view const& input,
                                 regex_program const& prog,
                                 size_type maxsplit,
                                 rmm::cuda_stream_view stream,
                                 rmm::mr::device_memory_resource* mr)
{
  return split_re(input, prog, split_direction::BACKWARD, maxsplit, stream, mr);
}

std::unique_ptr<column> rsplit_record_re(strings_column_view const& input,
                                         regex_program const& prog,
                                         size_type maxsplit,
                                         rmm::cuda_stream_view stream,
                                         rmm::mr::device_memory_resource* mr)
{
  return split_record_re(input, prog, split_direction::BACKWARD, maxsplit, stream, mr);
}

}  // namespace detail

// external APIs

std::unique_ptr<table> split_re(strings_column_view const& input,
                                std::string_view pattern,
                                size_type maxsplit,
                                rmm::mr::device_memory_resource* mr)
{
  CUDF_FUNC_RANGE();
  auto const h_prog = regex_program::create(pattern);
  return detail::split_re(input, *h_prog, maxsplit, cudf::get_default_stream(), mr);
}

std::unique_ptr<table> split_re(strings_column_view const& input,
                                regex_program const& prog,
                                size_type maxsplit,
                                rmm::mr::device_memory_resource* mr)
{
  CUDF_FUNC_RANGE();
  return detail::split_re(input, prog, maxsplit, cudf::get_default_stream(), mr);
}

std::unique_ptr<column> split_record_re(strings_column_view const& input,
                                        std::string_view pattern,
                                        size_type maxsplit,
                                        rmm::mr::device_memory_resource* mr)
{
  CUDF_FUNC_RANGE();
  auto const h_prog = regex_program::create(pattern);
  return detail::split_record_re(input, *h_prog, maxsplit, cudf::get_default_stream(), mr);
}

std::unique_ptr<column> split_record_re(strings_column_view const& input,
                                        regex_program const& prog,
                                        size_type maxsplit,
                                        rmm::mr::device_memory_resource* mr)
{
  CUDF_FUNC_RANGE();
  return detail::split_record_re(input, prog, maxsplit, cudf::get_default_stream(), mr);
}

std::unique_ptr<table> rsplit_re(strings_column_view const& input,
                                 std::string_view pattern,
                                 size_type maxsplit,
                                 rmm::mr::device_memory_resource* mr)
{
  CUDF_FUNC_RANGE();
  auto const h_prog = regex_program::create(pattern);
  return detail::rsplit_re(input, *h_prog, maxsplit, cudf::get_default_stream(), mr);
}

std::unique_ptr<table> rsplit_re(strings_column_view const& input,
                                 regex_program const& prog,
                                 size_type maxsplit,
                                 rmm::mr::device_memory_resource* mr)
{
  CUDF_FUNC_RANGE();
  return detail::rsplit_re(input, prog, maxsplit, cudf::get_default_stream(), mr);
}

std::unique_ptr<column> rsplit_record_re(strings_column_view const& input,
                                         std::string_view pattern,
                                         size_type maxsplit,
                                         rmm::mr::device_memory_resource* mr)
{
  CUDF_FUNC_RANGE();
  auto const h_prog = regex_program::create(pattern);
  return detail::rsplit_record_re(input, *h_prog, maxsplit, cudf::get_default_stream(), mr);
}

std::unique_ptr<column> rsplit_record_re(strings_column_view const& input,
                                         regex_program const& prog,
                                         size_type maxsplit,
                                         rmm::mr::device_memory_resource* mr)
{
  CUDF_FUNC_RANGE();
  return detail::rsplit_record_re(input, prog, maxsplit, cudf::get_default_stream(), mr);
}

}  // namespace strings
}  // namespace cudf
