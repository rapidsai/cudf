#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2023, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <text/utilities/tokenize_ops.cuh>

#include <nvtext/tokenize.hpp>

#include <cudf/column/column.hpp>
#include <cudf/column/column_device_view.cuh>
#include <cudf/column/column_factories.hpp>
#include <cudf/detail/get_value.cuh>
#include <cudf/detail/iterator.cuh>
#include <cudf/detail/null_mask.hpp>
#include <cudf/detail/nvtx/ranges.hpp>
#include <cudf/detail/sizes_to_offsets_iterator.cuh>
#include <cudf/detail/utilities/cuda.cuh>
#include <cudf/hashing/detail/hash_allocator.cuh>
#include <cudf/hashing/detail/murmurhash3_x86_32.cuh>
#include <cudf/strings/string_view.cuh>
#include <cudf/strings/strings_column_view.hpp>
#include <cudf/utilities/default_stream.hpp>
#include <cudf/utilities/error.hpp>

#include <rmm/cuda_stream_view.hpp>
#include <rmm/mr/device/polymorphic_allocator.hpp>

#include <cuco/static_map.cuh>

#include <thrust/copy.h>
#include <thrust/distance.h>
#include <thrust/execution_policy.h>
#include <thrust/functional.h>
#include <thrust/logical.h>
#include <thrust/transform.h>

#include <hipcub/hipcub.hpp>

namespace nvtext {
namespace detail {
namespace {

using string_hasher_type = cudf::hashing::detail::MurmurHash3_x86_32<cudf::string_view>;
using hash_value_type    = string_hasher_type::result_type;

/**
 * @brief Hasher function used for building and using the cuco static-map
 *
 * This takes advantage of heterogeneous lookup feature in cuco static-map which
 * allows inserting with one type (index) and looking up with a different type (string).
 */
struct vocab_hasher {
  cudf::column_device_view const d_strings;
  string_hasher_type hasher{};
  // used by insert
  __device__ hash_value_type operator()(cudf::size_type index) const
  {
    return hasher(d_strings.element<cudf::string_view>(index));
  }
  // used by find
  __device__ hash_value_type operator()(cudf::string_view const& s) const { return hasher(s); }
};

/**
 * @brief Equal function used for building and using the cuco static-map
 *
 * This takes advantage of heterogeneous lookup feature in cuco static-map which
 * allows inserting with one type (index) and looking up with a different type (string).
 */
struct vocab_equal {
  cudf::column_device_view const d_strings;
  // used by insert
  __device__ bool operator()(cudf::size_type lhs, cudf::size_type rhs) const noexcept
  {
    return lhs == rhs;  // all rows are expected to be unique
  }
  // used by find
  __device__ bool operator()(cudf::size_type lhs, cudf::string_view const& rhs) const noexcept
  {
    return d_strings.element<cudf::string_view>(lhs) == rhs;
  }
};

using hash_table_allocator_type = rmm::mr::stream_allocator_adaptor<default_allocator<char>>;
using probe_scheme              = cuco::experimental::linear_probing<1, vocab_hasher>;
using vocabulary_map_type       = cuco::experimental::static_map<cudf::size_type,
                                                           cudf::size_type,
                                                           cuco::experimental::extent<std::size_t>,
                                                           cuda::thread_scope_device,
                                                           vocab_equal,
                                                           probe_scheme,
                                                           hash_table_allocator_type>;
}  // namespace
}  // namespace detail

// since column_device_view::create returns is a little more than
// std::unique_ptr<column_device_view> this helper simplifies the return type in a maintainable way
using col_device_view = std::invoke_result_t<decltype(&cudf::column_device_view::create),
                                             cudf::column_view,
                                             rmm::cuda_stream_view>;

struct tokenize_vocabulary::tokenize_vocabulary_impl {
  std::unique_ptr<cudf::column> const vocabulary;
  col_device_view const d_vocabulary;
  std::unique_ptr<detail::vocabulary_map_type> vocabulary_map;

  auto get_map_ref() const { return vocabulary_map->ref(cuco::experimental::op::find); }

  tokenize_vocabulary_impl(std::unique_ptr<cudf::column>&& vocab,
                           col_device_view&& d_vocab,
                           std::unique_ptr<detail::vocabulary_map_type>&& map)
    : vocabulary(std::move(vocab)), d_vocabulary(std::move(d_vocab)), vocabulary_map(std::move(map))
  {
  }
};

struct key_pair {
  __device__ auto operator()(cudf::size_type idx) const noexcept
  {
    return cuco::make_pair(idx, idx);
  }
};

tokenize_vocabulary::tokenize_vocabulary(cudf::strings_column_view const& input,
                                         rmm::cuda_stream_view stream,
                                         rmm::mr::device_memory_resource* mr)
{
  CUDF_EXPECTS(not input.is_empty(), "vocabulary must not be empty");
  CUDF_EXPECTS(not input.has_nulls(), "vocabulary must not have nulls");

  // need to hold a copy of the input
  auto vocabulary   = std::make_unique<cudf::column>(input.parent(), stream, mr);
  auto d_vocabulary = cudf::column_device_view::create(vocabulary->view(), stream);

  auto vocab_map = std::make_unique<detail::vocabulary_map_type>(
    static_cast<size_t>(vocabulary->size() * 2),
    cuco::empty_key{-1},
    cuco::empty_value{-1},
    detail::vocab_equal{*d_vocabulary},
    detail::probe_scheme{detail::vocab_hasher{*d_vocabulary}},
    detail::hash_table_allocator_type{default_allocator<char>{}, stream},
    stream.value());

  // the row index is the token id (value for each key in the map)
  auto iter = cudf::detail::make_counting_transform_iterator(0, key_pair{});
  vocab_map->insert_async(iter, iter + vocabulary->size(), stream.value());

  _impl = new tokenize_vocabulary_impl(
    std::move(vocabulary), std::move(d_vocabulary), std::move(vocab_map));
}
tokenize_vocabulary::~tokenize_vocabulary() { delete _impl; }

std::unique_ptr<tokenize_vocabulary> load_vocabulary(cudf::strings_column_view const& input,
                                                     rmm::cuda_stream_view stream,
                                                     rmm::mr::device_memory_resource* mr)
{
  CUDF_FUNC_RANGE();
  return std::make_unique<tokenize_vocabulary>(input, stream, mr);
}

namespace detail {
namespace {

/**
 * @brief Threshold to decide on using string or warp parallel functions.
 *
 * If the average byte length of a string in a column exceeds this value then
 * the warp-parallel function is used to compute the output sizes.
 * Otherwise, a regular string-parallel function is used.
 *
 * This value was found using the vocab_tokenize benchmark results.
 */
constexpr cudf::size_type AVG_CHAR_BYTES_THRESHOLD = 128;

constexpr int block_size = 256;

__device__ bool is_delimiter(cudf::string_view const& d_delimiters, cudf::char_utf8 chr)
{
  return d_delimiters.empty() ? (chr <= ' ') :  // whitespace check
           thrust::any_of(thrust::seq,
                          d_delimiters.begin(),
                          d_delimiters.end(),
                          [chr] __device__(cudf::char_utf8 c) { return c == chr; });
}

struct mark_delimiters_fn {
  char const* d_chars;
  cudf::string_view const d_delimiter;
  int8_t* d_results;

  __device__ void operator()(cudf::size_type idx) const
  {
    auto const ptr = d_chars + idx;
    if (cudf::strings::detail::is_utf8_continuation_char(*ptr)) { return; }
    cudf::char_utf8 chr = 0;
    auto ch_size        = cudf::strings::detail::to_char_utf8(ptr, chr);
    auto const output   = is_delimiter(d_delimiter, chr);
    while (ch_size > 0) {
      d_results[idx++] = output;
      --ch_size;
    }
  }
};

__global__ void token_counts_fn(cudf::column_device_view const d_strings,
                                cudf::string_view const d_delimiter,
                                cudf::size_type* d_counts,
                                int8_t* d_results)
{
  // string per warp
  auto const idx = static_cast<std::size_t>(threadIdx.x + blockIdx.x * blockDim.x);
  if (idx >= (static_cast<std::size_t>(d_strings.size()) *
              static_cast<std::size_t>(cudf::detail::warp_size))) {
    return;
  }
  auto const str_idx  = static_cast<cudf::size_type>(idx / cudf::detail::warp_size);
  auto const lane_idx = static_cast<cudf::size_type>(idx % cudf::detail::warp_size);

  if (d_strings.is_null(str_idx)) {
    d_counts[str_idx] = 0;
    return;
  }
  auto const d_str = d_strings.element<cudf::string_view>(str_idx);
  if (d_str.empty()) {
    d_counts[str_idx] = 0;
    return;
  }

  auto const offsets =
    d_strings.child(cudf::strings_column_view::offsets_column_index).data<cudf::size_type>();
  auto const offset = offsets[str_idx + d_strings.offset()] - offsets[d_strings.offset()];
  auto const chars_begin =
    d_strings.child(cudf::strings_column_view::chars_column_index).data<char>() +
    offsets[d_strings.offset()];

  auto const begin        = d_str.data();
  auto const end          = begin + d_str.size_bytes();
  auto const d_output     = d_results + offset;
  auto const d_output_end = d_output + d_str.size_bytes();

  using warp_reduce = hipcub::WarpReduce<cudf::size_type>;
  __shared__ typename warp_reduce::TempStorage warp_storage;

  cudf::size_type count = 0;
  if (lane_idx == 0) {
    cudf::char_utf8 chr = 0;
    auto ch_size        = cudf::strings::detail::to_char_utf8(begin, chr);
    auto output         = 1;
    if (begin > chars_begin) {
      auto ptr = begin - 1;
      while (ptr > chars_begin && cudf::strings::detail::is_utf8_continuation_char(*ptr)) {
        --ptr;
      }
      cudf::strings::detail::to_char_utf8(ptr, chr);
      output = !is_delimiter(d_delimiter, chr);
    }
    auto ptr = d_output;
    while (ch_size > 0) {
      *ptr++ = output;
      --ch_size;
    }
    count = ((begin + ch_size) == end);
  }
  __syncwarp();

  for (auto itr = d_output + lane_idx + 1; itr < d_output_end; itr += cudf::detail::warp_size) {
    // add one if at the edge of a token or at the string's end
    count += ((*itr && !(*(itr - 1))) || (itr + 1 == d_output_end));
  }
  __syncwarp();

  // add up the counts from the other threads to compute the total token count for this string
  auto const total_count = warp_reduce(warp_storage).Reduce(count, hipcub::Sum());
  if (lane_idx == 0) { d_counts[str_idx] = total_count; }
}

/**
 * @brief Tokenizes each string and uses the map to assign token id values
 *
 * @tparam MapRefType Type of the static_map reference for calling find()
 */
template <typename MapRefType>
struct vocabulary_tokenizer_fn {
  cudf::column_device_view const d_strings;
  cudf::string_view const d_delimiter;
  MapRefType d_map;
  cudf::size_type const default_id;
  cudf::size_type const* d_offsets;
  cudf::size_type* d_results;

  __device__ void operator()(cudf::size_type idx) const
  {
    if (d_strings.is_null(idx)) { return; }

    auto const d_str = d_strings.element<cudf::string_view>(idx);
    characters_tokenizer tokenizer(d_str, d_delimiter);
    auto d_tokens = d_results + d_offsets[idx];

    cudf::size_type token_idx = 0;
    while (tokenizer.next_token()) {
      auto const pos   = tokenizer.token_byte_positions();
      auto const token = cudf::string_view{d_str.data() + pos.first, (pos.second - pos.first)};
      // lookup token in map
      auto const itr = d_map.find(token);
      auto const id  = (itr != d_map.end()) ? itr->second : default_id;
      // set value into the output
      d_tokens[token_idx++] = id;
    }
  }
};

template <typename MapRefType>
struct transform_tokenizer_fn {
  cudf::string_view const d_delimiter;
  MapRefType d_map;
  cudf::size_type const default_id;

  __device__ cudf::size_type operator()(cudf::string_view d_str) const
  {
    auto const begin = d_str.data();
    auto const end   = begin + d_str.size_bytes();

    auto itr = begin;
    while (itr < end) {
      cudf::char_utf8 chr = 0;
      auto const ch_size  = cudf::strings::detail::to_char_utf8(itr, chr);
      if (!is_delimiter(d_delimiter, chr)) break;
      itr += ch_size;
    }

    auto const size  = static_cast<cudf::size_type>(thrust::distance(itr, end));
    auto const token = cudf::string_view{itr, size};
    // lookup token in map
    auto const fitr = d_map.find(token);
    return (fitr != d_map.end()) ? fitr->second : default_id;
  }
};

}  // namespace

std::unique_ptr<cudf::column> tokenize_with_vocabulary(cudf::strings_column_view const& input,
                                                       tokenize_vocabulary const& vocabulary,
                                                       cudf::string_scalar const& delimiter,
                                                       cudf::size_type default_id,
                                                       rmm::cuda_stream_view stream,
                                                       rmm::mr::device_memory_resource* mr)
{
  CUDF_EXPECTS(delimiter.is_valid(stream), "Parameter delimiter must be valid");

  auto const output_type = cudf::data_type{cudf::type_to_id<cudf::size_type>()};
  if (input.size() == input.null_count()) { return cudf::make_empty_column(output_type); }

  // count the tokens per string and build the offsets from the counts
  auto const d_strings   = cudf::column_device_view::create(input.parent(), stream);
  auto const d_delimiter = delimiter.value(stream);
  auto map_ref           = vocabulary._impl->get_map_ref();
  auto const zero_itr    = thrust::make_counting_iterator<cudf::size_type>(0);

  if ((input.chars_size() / (input.size() - input.null_count())) < AVG_CHAR_BYTES_THRESHOLD) {
    auto const sizes_itr =
      cudf::detail::make_counting_transform_iterator(0, strings_tokenizer{*d_strings, d_delimiter});
    auto [token_offsets, total_count] =
      cudf::detail::make_offsets_child_column(sizes_itr, sizes_itr + input.size(), stream, mr);

    // build the output column to hold all the token ids
    auto tokens = cudf::make_numeric_column(
      output_type, total_count, cudf::mask_state::UNALLOCATED, stream, mr);
    auto d_tokens  = tokens->mutable_view().data<cudf::size_type>();
    auto d_offsets = token_offsets->view().data<cudf::size_type>();
    vocabulary_tokenizer_fn<decltype(map_ref)> tokenizer{
      *d_strings, d_delimiter, map_ref, default_id, d_offsets, d_tokens};
    thrust::for_each_n(rmm::exec_policy(stream), zero_itr, input.size(), tokenizer);
    return cudf::make_lists_column(input.size(),
                                   std::move(token_offsets),
                                   std::move(tokens),
                                   input.null_count(),
                                   cudf::detail::copy_bitmask(input.parent(), stream, mr),
                                   stream,
                                   mr);
  }

  // longer strings perform better with warp-parallel approach

  auto const first_offset  = (input.offset() == 0) ? 0
                                                   : cudf::detail::get_value<cudf::size_type>(
                                                      input.offsets(), input.offset(), stream);
  auto const last_offset   = (input.offset() == 0 && input.size() == input.offsets().size() - 1)
                               ? input.chars().size()
                               : cudf::detail::get_value<cudf::size_type>(
                                 input.offsets(), input.size() + input.offset(), stream);
  auto const chars_size    = last_offset - first_offset;
  auto const d_input_chars = input.chars().data<char>() + first_offset;

  rmm::device_uvector<cudf::size_type> d_token_counts(input.size(), stream);
  rmm::device_uvector<int8_t> d_marks(chars_size, stream);

  // mark position of all delimiters
  thrust::for_each_n(rmm::exec_policy(stream),
                     zero_itr,
                     chars_size,
                     mark_delimiters_fn{d_input_chars, d_delimiter, d_marks.data()});

  // launch warp per string to compute token counts
  cudf::detail::grid_1d grid{input.size() * cudf::detail::warp_size, block_size};
  token_counts_fn<<<grid.num_blocks, grid.num_threads_per_block, 0, stream.value()>>>(
    *d_strings, d_delimiter, d_token_counts.data(), d_marks.data());
  auto [token_offsets, total_count] = cudf::detail::make_offsets_child_column(
    d_token_counts.begin(), d_token_counts.end(), stream, mr);

  rmm::device_uvector<cudf::size_type> d_tmp_offsets(total_count + 1, stream);
  d_tmp_offsets.set_element(total_count, chars_size, stream);
  thrust::copy_if(rmm::exec_policy(stream),
                  zero_itr,
                  thrust::counting_iterator<cudf::size_type>(chars_size),
                  d_tmp_offsets.begin(),
                  [d_marks = d_marks.data()] __device__(auto idx) {
                    if (idx == 0) return true;
                    return d_marks[idx] && !d_marks[idx - 1];
                  });

  auto tmp_offsets =
    std::make_unique<cudf::column>(std::move(d_tmp_offsets), rmm::device_buffer{}, 0);
  auto tmp_chars = cudf::column_view(input.chars().type(), chars_size, d_input_chars, nullptr, 0);
  auto const tmp_input = cudf::column_view(
    input.parent().type(), total_count, nullptr, nullptr, 0, 0, {tmp_offsets->view(), tmp_chars});

  auto const d_tmp_strings = cudf::column_device_view::create(tmp_input, stream);

  auto tokens =
    cudf::make_numeric_column(output_type, total_count, cudf::mask_state::UNALLOCATED, stream, mr);
  auto d_tokens = tokens->mutable_view().data<cudf::size_type>();

  transform_tokenizer_fn<decltype(map_ref)> tokenizer{d_delimiter, map_ref, default_id};
  thrust::transform(rmm::exec_policy(stream),
                    d_tmp_strings->begin<cudf::string_view>(),
                    d_tmp_strings->end<cudf::string_view>(),
                    d_tokens,
                    tokenizer);

  return cudf::make_lists_column(input.size(),
                                 std::move(token_offsets),
                                 std::move(tokens),
                                 input.null_count(),
                                 cudf::detail::copy_bitmask(input.parent(), stream, mr),
                                 stream,
                                 mr);
}

}  // namespace detail

std::unique_ptr<cudf::column> tokenize_with_vocabulary(cudf::strings_column_view const& input,
                                                       tokenize_vocabulary const& vocabulary,
                                                       cudf::string_scalar const& delimiter,
                                                       cudf::size_type default_id,
                                                       rmm::cuda_stream_view stream,
                                                       rmm::mr::device_memory_resource* mr)
{
  CUDF_FUNC_RANGE();
  return detail::tokenize_with_vocabulary(input, vocabulary, delimiter, default_id, stream, mr);
}

}  // namespace nvtext
