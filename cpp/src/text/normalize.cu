#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2020-2025, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "text/normalize.cuh"
#include "text/subword/detail/data_normalizer.hpp"
#include "text/subword/detail/tokenizer_utils.cuh"
#include "text/utilities/tokenize_ops.cuh"

#include <cudf/column/column.hpp>
#include <cudf/column/column_device_view.cuh>
#include <cudf/column/column_factories.hpp>
#include <cudf/column/column_view.hpp>
#include <cudf/detail/iterator.cuh>
#include <cudf/detail/null_mask.hpp>
#include <cudf/detail/nvtx/ranges.hpp>
#include <cudf/sorting.hpp>
#include <cudf/strings/case.hpp>
#include <cudf/strings/detail/strings_children.cuh>
#include <cudf/strings/detail/strings_column_factories.cuh>
#include <cudf/strings/detail/utilities.cuh>
#include <cudf/strings/string_view.cuh>
#include <cudf/strings/strings_column_view.hpp>
#include <cudf/utilities/default_stream.hpp>
#include <cudf/utilities/memory_resource.hpp>

#include <nvtext/normalize.hpp>

#include <rmm/cuda_stream_view.hpp>

#include <hipcub/hipcub.hpp>
#include <cuda/functional>
#include <thrust/binary_search.h>
#include <thrust/execution_policy.h>
#include <thrust/for_each.h>
#include <thrust/functional.h>
#include <thrust/remove.h>
#include <thrust/transform_reduce.h>

#include <limits>

namespace nvtext {
namespace detail {
namespace {
/**
 * @brief Normalize spaces in a strings column.
 *
 * Repeated whitespace (code-point <= ' ') is replaced with a single space.
 * Also, whitespace is trimmed from the beginning and end of each string.
 *
 * This functor can be called to compute the output size in bytes
 * of each string and then called again to fill in the allocated buffer.
 */
struct normalize_spaces_fn {
  cudf::column_device_view const d_strings;  // strings to normalize
  cudf::size_type* d_sizes{};                // size of each output row
  char* d_chars{};                           // output buffer for characters
  cudf::detail::input_offsetalator d_offsets;

  __device__ void operator()(cudf::size_type idx)
  {
    if (d_strings.is_null(idx)) {
      if (!d_chars) { d_sizes[idx] = 0; }
      return;
    }
    cudf::string_view const single_space(" ", 1);
    auto const d_str = d_strings.element<cudf::string_view>(idx);
    char* buffer     = d_chars ? d_chars + d_offsets[idx] : nullptr;
    char* optr       = buffer;  // running output pointer

    cudf::size_type nbytes = 0;  // holds the number of bytes per output string

    // create a tokenizer for this string with whitespace delimiter (default)
    characters_tokenizer tokenizer(d_str);

    // this will retrieve tokens automatically skipping runs of whitespace
    while (tokenizer.next_token()) {
      auto const token_pos = tokenizer.token_byte_positions();
      auto const token =
        cudf::string_view(d_str.data() + token_pos.first, token_pos.second - token_pos.first);
      if (optr) {
        // prepend space unless we are at the beginning
        if (optr != buffer) { optr = cudf::strings::detail::copy_string(optr, single_space); }
        // write token to output buffer
        thrust::copy_n(thrust::seq, token.data(), token.size_bytes(), optr);
        optr += token.size_bytes();
      }
      nbytes += token.size_bytes() + 1;  // token size plus a single space
    }
    // remove trailing space
    if (!d_chars) { d_sizes[idx] = (nbytes > 0) ? nbytes - 1 : 0; }
  }
};

// code-point to multi-byte range limits
constexpr uint32_t UTF8_1BYTE = 0x0080;
constexpr uint32_t UTF8_2BYTE = 0x0800;
constexpr uint32_t UTF8_3BYTE = 0x01'0000;

__device__ int8_t cp_to_utf8(uint32_t codepoint, char* out)
{
  auto utf8 = cudf::strings::detail::codepoint_to_utf8(codepoint);
  return cudf::strings::detail::from_char_utf8(utf8, out);
}

/**
 * @brief Convert code-point arrays into UTF-8 bytes for each string.
 */
struct codepoint_to_utf8_fn {
  cudf::column_device_view const d_strings;  // input strings
  uint32_t const* cp_data;                   // full code-point array
  int64_t const* d_cp_offsets{};             // offsets to each string's code-point array
  cudf::size_type* d_sizes{};                // size of output string
  char* d_chars{};                           // buffer for the output strings column
  cudf::detail::input_offsetalator d_offsets;

  /**
   * @brief Return the number of bytes for the output string given its code-point array.
   *
   * @param str_cps code-points for the string
   * @param count number of code-points in `str_cps`
   * @return Number of bytes required for the output
   */
  __device__ cudf::size_type compute_output_size(uint32_t const* str_cps, uint32_t count)
  {
    return thrust::transform_reduce(
      thrust::seq,
      str_cps,
      str_cps + count,
      [](auto cp) { return 1 + (cp >= UTF8_1BYTE) + (cp >= UTF8_2BYTE) + (cp >= UTF8_3BYTE); },
      0,
      thrust::plus());
  }

  __device__ void operator()(cudf::size_type idx)
  {
    if (d_strings.is_null(idx)) {
      if (!d_chars) { d_sizes[idx] = 0; }
      return;
    }
    auto const offset = d_cp_offsets[idx];
    auto const count  = d_cp_offsets[idx + 1] - offset;  // number of code-points
    auto str_cps      = cp_data + offset;                // code-points for this string
    if (!d_chars) {
      d_sizes[idx] = compute_output_size(str_cps, count);
      return;
    }
    // convert each code-point to 1-4 UTF-8 encoded bytes
    char* out_ptr = d_chars + d_offsets[idx];
    for (uint32_t jdx = 0; jdx < count; ++jdx) {
      uint32_t codepoint = *str_cps++;
      out_ptr += cp_to_utf8(codepoint, out_ptr);
    }
  }
};

}  // namespace

// detail API
std::unique_ptr<cudf::column> normalize_spaces(cudf::strings_column_view const& strings,
                                               rmm::cuda_stream_view stream,
                                               rmm::device_async_resource_ref mr)
{
  if (strings.is_empty()) return cudf::make_empty_column(cudf::data_type{cudf::type_id::STRING});

  // create device column
  auto d_strings = cudf::column_device_view::create(strings.parent(), stream);

  // build offsets and children using the normalize_space_fn
  auto [offsets_column, chars] = cudf::strings::detail::make_strings_children(
    normalize_spaces_fn{*d_strings}, strings.size(), stream, mr);

  return cudf::make_strings_column(strings.size(),
                                   std::move(offsets_column),
                                   chars.release(),
                                   strings.null_count(),
                                   cudf::detail::copy_bitmask(strings.parent(), stream, mr));
}

/**
 * @copydoc nvtext::normalize_characters
 */
std::unique_ptr<cudf::column> normalize_characters(cudf::strings_column_view const& strings,
                                                   bool do_lower_case,
                                                   rmm::cuda_stream_view stream,
                                                   rmm::device_async_resource_ref mr)
{
  if (strings.is_empty()) return cudf::make_empty_column(cudf::data_type{cudf::type_id::STRING});

  // create the normalizer and call it
  auto result = [&] {
    auto const cp_metadata = get_codepoint_metadata(stream);
    auto const aux_table   = get_aux_codepoint_data(stream);
    auto const normalizer  = data_normalizer(cp_metadata.data(), aux_table.data(), do_lower_case);
    return normalizer.normalize(strings, stream);
  }();

  CUDF_EXPECTS(
    result.first->size() < static_cast<std::size_t>(std::numeric_limits<cudf::size_type>::max()),
    "output exceeds the column size limit",
    std::overflow_error);

  // convert the result into a strings column
  // - the cp_chars are the new 4-byte code-point values for all the characters in the output
  // - the cp_offsets identify which code-points go with which strings
  auto const cp_chars   = result.first->data();
  auto const cp_offsets = result.second->data();

  auto d_strings = cudf::column_device_view::create(strings.parent(), stream);

  // build offsets and children using the codepoint_to_utf8_fn
  auto [offsets_column, chars] = cudf::strings::detail::make_strings_children(
    codepoint_to_utf8_fn{*d_strings, cp_chars, cp_offsets}, strings.size(), stream, mr);

  return cudf::make_strings_column(strings.size(),
                                   std::move(offsets_column),
                                   chars.release(),
                                   strings.null_count(),
                                   cudf::detail::copy_bitmask(strings.parent(), stream, mr));
}

}  // namespace detail

// external APIs

std::unique_ptr<cudf::column> normalize_spaces(cudf::strings_column_view const& input,
                                               rmm::cuda_stream_view stream,
                                               rmm::device_async_resource_ref mr)
{
  CUDF_FUNC_RANGE();
  return detail::normalize_spaces(input, stream, mr);
}

/**
 * @copydoc nvtext::normalize_characters
 */
std::unique_ptr<cudf::column> normalize_characters(cudf::strings_column_view const& input,
                                                   bool do_lower_case,
                                                   rmm::cuda_stream_view stream,
                                                   rmm::device_async_resource_ref mr)
{
  CUDF_FUNC_RANGE();
  return detail::normalize_characters(input, do_lower_case, stream, mr);
}

struct character_normalizer::character_normalizer_impl {
  rmm::device_uvector<uint32_t> cp_metadata;
  rmm::device_uvector<aux_codepoint_data_type> aux_table;
  bool do_lower_case;
  std::unique_ptr<cudf::column> special_tokens;
  rmm::device_uvector<cudf::string_view> special_tokens_view;

  cudf::device_span<cudf::string_view const> get_special_tokens() const
  {
    return special_tokens_view;
  }

  character_normalizer_impl(rmm::device_uvector<uint32_t>&& cp_metadata,
                            rmm::device_uvector<aux_codepoint_data_type>&& aux_table,
                            bool do_lower_case,
                            std::unique_ptr<cudf::column>&& special_tokens,
                            rmm::device_uvector<cudf::string_view>&& special_tokens_view)
    : cp_metadata(std::move(cp_metadata)),
      aux_table(std::move(aux_table)),
      do_lower_case{do_lower_case},
      special_tokens{std::move(special_tokens)},
      special_tokens_view{std::move(special_tokens_view)}
  {
  }
};

character_normalizer::character_normalizer(bool do_lower_case,
                                           cudf::strings_column_view const& special_tokens,
                                           rmm::cuda_stream_view stream,
                                           rmm::device_async_resource_ref)
{
  auto cp_metadata = nvtext::detail::get_codepoint_metadata(stream);
  auto aux_table   = nvtext::detail::get_aux_codepoint_data(stream);
  CUDF_EXPECTS(
    !special_tokens.has_nulls(), "special tokens should not have nulls", std::invalid_argument);

  auto sorted = std::move(
    cudf::sort(cudf::table_view({special_tokens.parent()}), {}, {}, stream)->release().front());
  if (do_lower_case) {
    // lower-case the tokens so they will match the normalized input
    sorted = cudf::strings::to_lower(cudf::strings_column_view(sorted->view()), stream);
  }

  auto tokens_view = cudf::strings::detail::create_string_vector_from_column(
    cudf::strings_column_view(sorted->view()), stream, cudf::get_current_device_resource_ref());

  _impl = std::make_unique<character_normalizer_impl>(std::move(cp_metadata),
                                                      std::move(aux_table),
                                                      do_lower_case,
                                                      std::move(sorted),
                                                      std::move(tokens_view));
}

character_normalizer::~character_normalizer() {}

std::unique_ptr<character_normalizer> create_character_normalizer(
  bool do_lower_case,
  cudf::strings_column_view const& special_tokens,
  rmm::cuda_stream_view stream,
  rmm::device_async_resource_ref mr)
{
  CUDF_FUNC_RANGE();
  return std::make_unique<character_normalizer>(do_lower_case, special_tokens, stream, mr);
}

namespace detail {
namespace {

/**
 * @brief Kernel handles fixing up the normalized data to account for any special tokens
 *
 * This undoes the padding added around the `[]` for patterns matching the strings in the
 * special_tokens array.
 *
 * Launched as a thread per input byte (total_count).
 *
 * @param d_normalized The normalized set of UTF-8 characters; 3 uints per input byte
 * @param total_count Number of bytes represented by d_normalized; len(d_normalized)/3
 * @param special_tokens Tokens to check against
 */
CUDF_KERNEL void special_tokens_kernel(uint32_t* d_normalized,
                                       int64_t total_count,
                                       cudf::device_span<cudf::string_view const> special_tokens)
{
  auto const idx = cudf::detail::grid_1d::global_thread_id();
  if (idx >= total_count) { return; }
  auto const begin = d_normalized + (idx * MAX_NEW_CHARS) + 1;
  if (*begin != '[') { return; }
  auto const end   = begin + cuda::std::min(6L, total_count - idx) * MAX_NEW_CHARS;
  auto const match = thrust::find(thrust::seq, begin, end, static_cast<uint32_t>(']'));
  if (match == end) { return; }
  char candidate[8];
  auto const ch_begin =
    thrust::transform_iterator(begin, [](auto v) { return static_cast<char>(v); });
  auto const ch_end = ch_begin + thrust::distance(begin, match + 1);
  auto last         = thrust::copy_if(
    thrust::seq, ch_begin, ch_end, candidate, [](auto c) { return c != 0 && c != ' '; });
  *last = 0;  // only needed for debug

  auto const size  = static_cast<cudf::size_type>(thrust::distance(candidate, last));
  auto const token = cudf::string_view(candidate, size);
  // the binary_search expects the special_tokens to be sorted
  if (!thrust::binary_search(thrust::seq, special_tokens.begin(), special_tokens.end(), token)) {
    return;
  }

  // fix up chars to remove the extra spaces
  *(begin + 1) = 0;  // removes space after '['
  *(match - 1) = 0;  // removes space before ']'
}

/**
 * @brief The normalizer kernel
 *
 * Launched as a thread per input byte (total_bytes).
 *
 * Converts the input d_chars into codepoints to lookup in the provided tables.
 * Once processed, the d_output contains 3 uints per input byte each encoded
 * as output UTF-8. Any zero values are to removed by a subsequent kernel call.
 *
 * @param d_chars The characters for the input strings column to normalize
 * @param total_bytes The number of bytes in the d_chars
 * @param cp_metadata First lookup table for codepoint metadata
 * @param aux_table Second lookup table containing possible replacement characters
 * @param do_lower_case True if the normalization includes lower-casing characters
 * @param d_output The output of the normalization (UTF-8 encoded)
 */
CUDF_KERNEL void data_normalizer_kernel(char const* d_chars,
                                        int64_t total_bytes,
                                        codepoint_metadata_type const* cp_metadata,
                                        aux_codepoint_data_type const* aux_table,
                                        bool do_lower_case,
                                        uint32_t* d_output)
{
  uint32_t replacement[MAX_NEW_CHARS] = {0};

  auto const idx = cudf::detail::grid_1d::global_thread_id();

  if ((idx < total_bytes) && cudf::strings::detail::is_begin_utf8_char(d_chars[idx])) {
    auto const cp = [utf8 = d_chars + idx] {
      cudf::char_utf8 ch_utf8 = *utf8;
      if (ch_utf8 > 0x7F) { cudf::strings::detail::to_char_utf8(utf8, ch_utf8); }
      return cudf::strings::detail::utf8_to_codepoint(ch_utf8);
    }();
    auto const metadata = cp_metadata[cp];

    if (!should_remove_cp(metadata, do_lower_case)) {
      int8_t num_new_chars = 1;
      // retrieve the normalized value for cp
      auto const new_cp = do_lower_case || always_replace(metadata) ? get_first_cp(metadata) : cp;
      replacement[0]    = new_cp == 0 ? cp : new_cp;

      if (do_lower_case && is_multi_char_transform(metadata)) {
        auto const next_cps = aux_table[cp];
        replacement[1]      = static_cast<uint32_t>(next_cps >> 32);
        replacement[2]      = static_cast<uint32_t>(next_cps & 0xFFFFFFFF);
        num_new_chars       = 2 + (replacement[2] != 0);
      }

      if (should_add_spaces(metadata, do_lower_case) && (num_new_chars == 1)) {
        replacement[1] = replacement[0];
        replacement[0] = SPACE_CODE_POINT;  // add spaces around the new codepoint
        replacement[2] = SPACE_CODE_POINT;
        num_new_chars  = 3;
      }

      // convert codepoints back to UTF-8 in-place
      for (int k = 0; k < num_new_chars; ++k) {
        auto const new_cp = replacement[k];
        if (new_cp) { cp_to_utf8(new_cp, reinterpret_cast<char*>(replacement + k)); }
      }
    }
  }

  // employ an optimized coalesced writer to output replacement as a block of transposed data
  using block_store =
    hipcub::BlockStore<uint32_t, 256, MAX_NEW_CHARS, hipcub::BLOCK_STORE_WARP_TRANSPOSE>;
  __shared__ typename block_store::TempStorage bs_stg;
  auto block_base = d_output + blockIdx.x * blockDim.x * MAX_NEW_CHARS;
  block_store(bs_stg).Store(block_base, replacement);
}

/**
 * @brief Computes the output sizes for each row
 *
 * The input offsets are used with segmented-reduce to count the number of
 * non-zero values for each output row.
 *
 * @param d_normalized The UTF-8 encoded normalized values
 * @param offsets These identify the row boundaries
 * @param offset Only non-zero if the input column has been sliced
 * @param size The number of output rows (sames as the number of input rows)
 * @param stream Stream used for allocating device memory and launching kernels
 * @return The sizes of each output row
 */
template <typename OffsetType>
rmm::device_uvector<cudf::size_type> compute_sizes(cudf::device_span<uint32_t const> d_normalized,
                                                   OffsetType offsets,
                                                   int64_t offset,
                                                   cudf::size_type size,
                                                   rmm::cuda_stream_view stream)
{
  auto output_sizes = rmm::device_uvector<cudf::size_type>(size, stream);

  auto d_data = d_normalized.data();

  // counts the non-zero bytes in the d_data array
  auto d_in = cudf::detail::make_counting_transform_iterator(
    0, cuda::proclaim_return_type<cudf::size_type>([d_data] __device__(auto idx) {
      idx = idx * MAX_NEW_CHARS;
      // transform function counts number of non-zero bytes in uint32_t value
      auto tfn = [](uint32_t v) -> cudf::size_type {
        return ((v & 0xFF) > 0) + ((v & 0xFF00) > 0) + ((v & 0xFF0000) > 0) +
               ((v & 0xFF000000) > 0);
      };
      auto const begin = d_data + idx;
      auto const end   = begin + MAX_NEW_CHARS;
      return thrust::transform_reduce(thrust::seq, begin, end, tfn, 0, thrust::plus{});
    }));

  // DeviceSegmentedReduce is used to compute the size of each output row
  auto d_out = output_sizes.begin();
  auto temp  = std::size_t{0};
  if (offset == 0) {
    hipcub::DeviceSegmentedReduce::Sum(
      nullptr, temp, d_in, d_out, size, offsets, offsets + 1, stream.value());
    auto d_temp = rmm::device_buffer{temp, stream};
    hipcub::DeviceSegmentedReduce::Sum(
      d_temp.data(), temp, d_in, d_out, size, offsets, offsets + 1, stream.value());
  } else {
    // offsets need to be normalized for segmented-reduce to work efficiently
    auto offsets_itr = thrust::transform_iterator(
      offsets,
      cuda::proclaim_return_type<int64_t>([offset] __device__(auto o) { return o - offset; }));
    hipcub::DeviceSegmentedReduce::Sum(
      nullptr, temp, d_in, d_out, size, offsets_itr, offsets_itr + 1, stream.value());
    auto d_temp = rmm::device_buffer{temp, stream};
    hipcub::DeviceSegmentedReduce::Sum(
      d_temp.data(), temp, d_in, d_out, size, offsets_itr, offsets_itr + 1, stream.value());
  }

  return output_sizes;
}

// handles ranges above int32 max
template <typename InputIterator, typename OutputIterator, typename T>
OutputIterator remove_copy_safe(InputIterator first,
                                InputIterator last,
                                OutputIterator result,
                                T const& value,
                                rmm::cuda_stream_view stream)
{
  auto const copy_size = std::min(static_cast<std::size_t>(std::distance(first, last)),
                                  static_cast<std::size_t>(std::numeric_limits<int>::max()));

  auto itr = first;
  while (itr != last) {
    auto const copy_end =
      static_cast<std::size_t>(std::distance(itr, last)) <= copy_size ? last : itr + copy_size;
    result = thrust::remove_copy(rmm::exec_policy(stream), itr, copy_end, result, value);
    itr    = copy_end;
  }
  return result;
}

// handles ranges above int32 max
template <typename Iterator, typename T>
Iterator remove_safe(Iterator first, Iterator last, T const& value, rmm::cuda_stream_view stream)
{
  auto const size = std::min(static_cast<std::size_t>(std::distance(first, last)),
                             static_cast<std::size_t>(std::numeric_limits<int>::max()));

  auto result = first;
  auto itr    = first;
  while (itr != last) {
    auto end = static_cast<std::size_t>(std::distance(itr, last)) <= size ? last : itr + size;
    result   = thrust::remove(rmm::exec_policy(stream), itr, end, value);
    itr      = end;
  }
  return result;
}
}  // namespace

std::unique_ptr<cudf::column> normalize_characters(cudf::strings_column_view const& input,
                                                   character_normalizer const& normalizer,
                                                   rmm::cuda_stream_view stream,
                                                   rmm::device_async_resource_ref mr)
{
  if (input.is_empty()) { return cudf::make_empty_column(cudf::data_type{cudf::type_id::STRING}); }

  auto [first_offset, last_offset] =
    cudf::strings::detail::get_first_and_last_offset(input, stream);
  auto const chars_size    = last_offset - first_offset;
  auto const d_input_chars = input.chars_begin(stream) + first_offset;

  if (chars_size == 0) { return std::make_unique<cudf::column>(input.parent(), stream, mr); }

  constexpr int64_t block_size = 256;
  cudf::detail::grid_1d grid{chars_size, block_size};
  auto const max_new_char_total = cudf::util::round_up_safe(chars_size, block_size) * MAX_NEW_CHARS;

  auto const& parameters = normalizer._impl;

  auto d_normalized = rmm::device_uvector<uint32_t>(max_new_char_total, stream);
  data_normalizer_kernel<<<grid.num_blocks, grid.num_threads_per_block, 0, stream.value()>>>(
    d_input_chars,
    chars_size,
    parameters->cp_metadata.data(),
    parameters->aux_table.data(),
    parameters->do_lower_case,
    d_normalized.data());

  // This removes space added around any special tokens in the form of [ttt].
  // An alternate approach is to do a multi-replace of '[ ttt ]' with '[ttt]' right
  // before returning the output strings column.
  auto const special_tokens = parameters->get_special_tokens();
  if (!special_tokens.empty()) {
    special_tokens_kernel<<<grid.num_blocks, grid.num_threads_per_block, 0, stream.value()>>>(
      d_normalized.data(), chars_size, special_tokens);
  }

  // Use segmented-reduce over the non-zero codepoints to get the size of the output rows
  auto const input_offsets =
    cudf::detail::offsetalator_factory::make_input_iterator(input.offsets(), input.offset());
  auto output_sizes =
    compute_sizes(d_normalized, input_offsets, first_offset, input.size(), stream);

  // convert the sizes to offsets
  auto [offsets, total_size] = cudf::strings::detail::make_offsets_child_column(
    output_sizes.begin(), output_sizes.end(), stream, mr);

  // create output chars by calling remove_copy(0) on the bytes in d_normalized
  auto chars       = rmm::device_uvector<char>(total_size, stream, mr);
  auto const begin = reinterpret_cast<char const*>(d_normalized.begin());
  // the remove() above speeds up the remove_copy() by roughly 10%
  auto const end =
    reinterpret_cast<char const*>(remove_safe(d_normalized.begin(), d_normalized.end(), 0, stream));
  remove_copy_safe(begin, end, chars.data(), 0, stream);

  return cudf::make_strings_column(input.size(),
                                   std::move(offsets),
                                   chars.release(),
                                   input.null_count(),
                                   cudf::detail::copy_bitmask(input.parent(), stream, mr));
}

}  // namespace detail

std::unique_ptr<cudf::column> normalize_characters(cudf::strings_column_view const& input,
                                                   character_normalizer const& normalizer,
                                                   rmm::cuda_stream_view stream,
                                                   rmm::device_async_resource_ref mr)
{
  CUDF_FUNC_RANGE();
  return detail::normalize_characters(input, normalizer, stream, mr);
}

}  // namespace nvtext
