#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2025, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <cudf/column/column.hpp>
#include <cudf/column/column_device_view.cuh>
#include <cudf/column/column_factories.hpp>
#include <cudf/detail/nvtx/ranges.hpp>
#include <cudf/strings/detail/strings_column_factories.cuh>
#include <cudf/strings/detail/utilities.hpp>
#include <cudf/strings/string_view.cuh>
#include <cudf/strings/strings_column_view.hpp>
#include <cudf/utilities/default_stream.hpp>
#include <cudf/utilities/error.hpp>
#include <cudf/utilities/memory_resource.hpp>

#include <nvtext/deduplicate.hpp>

#include <rmm/cuda_stream_view.hpp>
#include <rmm/device_uvector.hpp>

#include <hipcub/hipcub.hpp>
#include <cuda/std/functional>
#include <cuda/std/limits>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/remove.h>
#include <thrust/sort.h>
#include <thrust/transform.h>
#include <thrust/unique.h>

namespace nvtext {
namespace detail {
namespace {

using string_index = cudf::strings::detail::string_index_pair;

/**
 * @brief Comparator used for building the suffix array
 *
 * Each string is created using the substring of the chars_span
 * between the given index/offset (lhs or rhs) and the end of the
 * chars_span.
 */
struct sort_comparator_fn {
  cudf::device_span<char const> chars_span;
  __device__ bool operator()(cudf::size_type lhs, cudf::size_type rhs) const
  {
    auto const chars_size = static_cast<cudf::size_type>(chars_span.size());
    auto const lh_str     = cudf::string_view(chars_span.data() + lhs, chars_size - lhs);
    auto const rh_str     = cudf::string_view(chars_span.data() + rhs, chars_size - rhs);
    return lh_str < rh_str;
  }
};

/**
 * @brief Utility counts the number of common bytes between the 2 given strings
 */
__device__ cudf::size_type common_prefix_length(cudf::string_view lhs, cudf::string_view rhs)
{
  auto const size1 = lhs.size_bytes();
  auto const size2 = rhs.size_bytes();
  auto const* ptr1 = lhs.data();
  auto const* ptr2 = rhs.data();

  cudf::size_type idx = 0;
  while ((idx < size1) && (idx < size2) && (*ptr1++ == *ptr2++)) {
    ++idx;
  }
  return idx;
}

/**
 * @brief Uses the sorted array of indices (suffix array) to compare common
 * bytes between adjacent strings and return the count if greater than width.
 */
struct find_adjacent_duplicates_fn {
  cudf::device_span<char const> chars_span;
  cudf::size_type width;
  cudf::size_type const* d_indices;
  __device__ int16_t operator()(cudf::size_type idx) const
  {
    if (idx == 0) { return 0; }
    auto const chars_size = static_cast<cudf::size_type>(chars_span.size());

    auto const lhs    = d_indices[idx - 1];
    auto const rhs    = d_indices[idx];
    auto const lh_str = cudf::string_view(chars_span.data() + lhs, chars_size - lhs);
    auto const rh_str = cudf::string_view(chars_span.data() + rhs, chars_size - rhs);

    constexpr auto max_common_length =
      static_cast<cudf::size_type>(cuda::std::numeric_limits<int16_t>::max());

    auto const size = cuda::std::min(common_prefix_length(lh_str, rh_str), max_common_length);

    return size >= width ? static_cast<int16_t>(size) : 0;
  }
};

/**
 * @brief Resolves any overlapping duplicate strings found in find_adjacent_duplicates_fn
 *
 * Adjacent strings with common bytes greater than width may produce a sliding window of
 * duplicates which can be collapsed into a single duplicate pattern instead.
 * Care is also taken to preserve adjacent strings which may be larger than max(int16).
 */
struct collapse_overlaps_fn {
  char const* d_chars;
  cudf::size_type const* d_suffix_offsets;
  int16_t const* d_sizes;
  __device__ string_index operator()(cudf::size_type idx) const
  {
    constexpr auto max_common_length =
      static_cast<cudf::size_type>(cuda::std::numeric_limits<int16_t>::max());

    auto size   = d_sizes[idx];
    auto offset = d_suffix_offsets[idx];
    if ((idx > 0) && ((offset - 1) == d_suffix_offsets[idx - 1])) {
      if (size < d_sizes[idx - 1]) { return string_index{nullptr, 0}; }
      if (size == d_sizes[idx - 1] && size == max_common_length) {
        // check if we are in the middle of a chain
        auto prev_idx    = idx - max_common_length;
        auto prev_offset = offset;
        while (prev_idx >= 0) {
          if (d_suffix_offsets[prev_idx] != (prev_offset - max_common_length)) {
            prev_idx = -1;
          } else {
            if (d_sizes[idx + 1] < size) { break; }  // final edge
            prev_offset = d_suffix_offsets[prev_idx];
            if ((prev_idx == 0) || ((prev_offset - 1) != d_suffix_offsets[prev_idx - 1])) { break; }
            prev_idx -= max_common_length;
          }
        }
        if (prev_idx < 0) { return string_index{nullptr, 0}; }
      }
    }

    auto d_ptr = d_chars + offset;
    return string_index(d_ptr, size);
  }
};

std::unique_ptr<rmm::device_uvector<cudf::size_type>> build_suffix_array_fn(
  cudf::device_span<char const> chars_span,
  cudf::size_type min_width,
  rmm::cuda_stream_view stream,
  rmm::device_async_resource_ref mr)
{
  auto const size = static_cast<cudf::size_type>(chars_span.size()) - min_width + (min_width > 0);
  auto indices    = rmm::device_uvector<cudf::size_type>(size, stream);

  auto const cmp_op = sort_comparator_fn{chars_span};
  auto const seq    = thrust::make_counting_iterator<cudf::size_type>(0);
  auto tmp_bytes    = std::size_t{0};
  hipcub::DeviceMergeSort::SortKeysCopy(
    nullptr, tmp_bytes, seq, indices.begin(), indices.size(), cmp_op, stream.value());
  auto tmp_stg = rmm::device_buffer(tmp_bytes, stream);
  hipcub::DeviceMergeSort::SortKeysCopy(
    tmp_stg.data(), tmp_bytes, seq, indices.begin(), indices.size(), cmp_op, stream.value());

  return std::make_unique<rmm::device_uvector<cudf::size_type>>(std::move(indices));
}

std::unique_ptr<cudf::column> resolve_duplicates_fn(
  cudf::device_span<char const> chars_span,
  cudf::device_span<cudf::size_type const> indices,
  cudf::size_type min_width,
  rmm::cuda_stream_view stream,
  rmm::device_async_resource_ref mr)
{
  auto sizes = rmm::device_uvector<int16_t>(indices.size(), stream);

  // locate candidate duplicates within the suffix array
  thrust::transform(rmm::exec_policy_nosync(stream),
                    thrust::counting_iterator<cudf::size_type>(0),
                    thrust::counting_iterator<cudf::size_type>(indices.size()),
                    sizes.begin(),
                    find_adjacent_duplicates_fn{chars_span, min_width, indices.data()});

  auto const dup_count =
    sizes.size() - thrust::count(rmm::exec_policy(stream), sizes.begin(), sizes.end(), 0);
  auto dup_indices = rmm::device_uvector<cudf::size_type>(dup_count, stream);

  // remove the non-candidate entries from indices and sizes
  thrust::remove_copy_if(
    rmm::exec_policy(stream),
    indices.begin(),
    indices.end(),
    thrust::counting_iterator<cudf::size_type>(0),
    dup_indices.begin(),
    [d_sizes = sizes.data()] __device__(cudf::size_type idx) -> bool { return d_sizes[idx] == 0; });
  auto end = thrust::remove(rmm::exec_policy(stream), sizes.begin(), sizes.end(), 0);
  sizes.resize(cuda::std::distance(sizes.begin(), end), stream);

  // sort the resulting indices/sizes for overlap filtering
  thrust::sort_by_key(
    rmm::exec_policy_nosync(stream), dup_indices.begin(), dup_indices.end(), sizes.begin());

  // produce final duplicates for make_strings_column and collapse any overlapping candidates
  auto duplicates =
    rmm::device_uvector<cudf::strings::detail::string_index_pair>(dup_count, stream);
  thrust::transform(rmm::exec_policy_nosync(stream),
                    thrust::counting_iterator<cudf::size_type>(0),
                    thrust::counting_iterator<cudf::size_type>(dup_indices.size()),
                    duplicates.begin(),
                    collapse_overlaps_fn{chars_span.data(), dup_indices.data(), sizes.data()});

  // filter out the remaining non-viable candidates
  duplicates.resize(
    cuda::std::distance(
      duplicates.begin(),
      thrust::remove(
        rmm::exec_policy(stream), duplicates.begin(), duplicates.end(), string_index{nullptr, 0})),
    stream);

  // sort the result by size descending (should be very fast)
  thrust::sort(rmm::exec_policy_nosync(stream),
               duplicates.begin(),
               duplicates.end(),
               [] __device__(auto lhs, auto rhs) -> bool { return lhs.second > rhs.second; });

  // ironically remove duplicates from the sorted list
  duplicates.resize(
    cuda::std::distance(duplicates.begin(),
                        thrust::unique(rmm::exec_policy(stream),
                                       duplicates.begin(),
                                       duplicates.end(),
                                       [] __device__(auto lhs, auto rhs) -> bool {
                                         return cudf::string_view(lhs.first, lhs.second) ==
                                                cudf::string_view(rhs.first, rhs.second);
                                       })),
    stream);

  return cudf::strings::detail::make_strings_column(
    duplicates.begin(), duplicates.end(), stream, mr);
}

}  // namespace

std::unique_ptr<rmm::device_uvector<cudf::size_type>> build_suffix_array(
  cudf::strings_column_view const& input,
  cudf::size_type min_width,
  rmm::cuda_stream_view stream,
  rmm::device_async_resource_ref mr)
{
  auto [first_offset, last_offset] =
    cudf::strings::detail::get_first_and_last_offset(input, stream);

  auto const d_input_chars = input.chars_begin(stream) + first_offset;
  auto const chars_size    = last_offset - first_offset;
  CUDF_EXPECTS(
    min_width < chars_size, "min_width value cannot exceed the input size", std::invalid_argument);
  CUDF_EXPECTS(chars_size < std::numeric_limits<cudf::size_type>::max(),
               "Input size cannot exceed the maximum integer size limit of 2GB",
               std::invalid_argument);

  auto const chars_span = cudf::device_span<char const>(d_input_chars, chars_size);
  return build_suffix_array_fn(chars_span, min_width, stream, mr);
}

std::unique_ptr<cudf::column> resolve_duplicates(cudf::strings_column_view const& input,
                                                 cudf::device_span<cudf::size_type const> indices,
                                                 cudf::size_type min_width,
                                                 rmm::cuda_stream_view stream,
                                                 rmm::device_async_resource_ref mr)
{
  CUDF_EXPECTS(min_width > 8, "min_width should be at least 8", std::invalid_argument);

  auto d_strings = cudf::column_device_view::create(input.parent(), stream);
  auto [first_offset, last_offset] =
    cudf::strings::detail::get_first_and_last_offset(input, stream);
  auto const d_input_chars = input.chars_begin(stream) + first_offset;
  auto const chars_size    = last_offset - first_offset;
  CUDF_EXPECTS(
    min_width < chars_size, "min_width value cannot exceed the input size", std::invalid_argument);
  CUDF_EXPECTS(chars_size < std::numeric_limits<cudf::size_type>::max(),
               "Input size cannot exceed the maximum integer size limit of 2GB",
               std::invalid_argument);

  auto const chars_span = cudf::device_span<char const>(d_input_chars, chars_size);

  return resolve_duplicates_fn(chars_span, indices, min_width, stream, mr);
}

namespace {

/**
 * @brief Resolves duplicates between two strings chars
 *
 * Searches chars_span2 for each string in chars_span1 (idx) using the sorted array
 * indices for both. The d_indices1 provides the target string while d_indices2 is
 * searched for a common string.
 *
 * The search uses lower-bound to locate a common string but limited to the prefix search
 * output provided in the lb_indices/ub_indices. This improves the search by 2x over
 * executing lower-bound over the entire sorted indices.
 *
 * Returns the size of the common bytes found only if greater than width.
 */
struct find_duplicates_fn {
  cudf::device_span<char const> chars_span1;
  cudf::device_span<char const> chars_span2;
  cudf::size_type width;
  cudf::device_span<cudf::size_type const> d_indices1;
  cudf::device_span<cudf::size_type const> d_indices2;
  cudf::device_span<cudf::size_type const> lb_indices;
  cudf::device_span<cudf::size_type const> ub_indices;

  __device__ int16_t operator()(cudf::size_type idx) const
  {
    auto const lhs_idx  = d_indices1[idx];
    auto const lhs_size = static_cast<cudf::size_type>(chars_span1.size() - lhs_idx);
    auto const lh_str   = cudf::string_view(chars_span1.data() + lhs_idx, lhs_size);

    auto const itr = cudf::detail::make_counting_transform_iterator(
      0, [d_chars = chars_span2.data(), d_indices = d_indices2, cs = chars_span2.size()](auto idx) {
        auto const rhs_idx  = d_indices[idx];
        auto const rhs_size = static_cast<cudf::size_type>(cs - rhs_idx);
        return cudf::string_view(d_chars + rhs_idx, rhs_size);
      });
    auto const lb_idx = lb_indices[idx];
    auto const ub_idx = ub_indices[idx];
    auto const begin  = itr + lb_idx;
    auto const end    = itr + ub_idx + (ub_idx < d_indices2.size());

    auto const fnd = thrust::lower_bound(thrust::seq, begin, end, lh_str);
    if (fnd == end) { return 0; }
    auto size = common_prefix_length(lh_str, *fnd);

    // check for lower_bound edge case
    auto const ridx = lb_idx + cuda::std::distance(begin, fnd) - (lb_idx > 0);
    if (size < width && ridx >= 0) {
      auto const rhs_idx  = d_indices2[ridx];
      auto const rhs_size = static_cast<cudf::size_type>(chars_span2.size() - rhs_idx);
      auto const rh_str   = cudf::string_view(chars_span2.data() + rhs_idx, rhs_size);

      size = common_prefix_length(lh_str, rh_str);
    }

    constexpr auto max_common_length =
      static_cast<cudf::size_type>(cuda::std::numeric_limits<int16_t>::max());

    size = cuda::std::min(size, max_common_length);
    return size >= width ? static_cast<int16_t>(size) : 0;
  }
};

/**
 * @brief Builds a string_view from a chars index
 */
struct index_to_string_fn {
  cudf::device_span<char const> chars;
  __device__ cudf::string_view operator()(cudf::size_type idx) const
  {
    auto const size = static_cast<cudf::size_type>(chars.size() - idx);
    return cudf::string_view(chars.data() + idx, size);
  }
};

/**
 * @brief Builds a prefix string from a string_view
 */
struct string_to_prefix_fn {
  __device__ uint32_t operator()(cudf::string_view str) const
  {
    uint32_t data   = 0;
    auto const size = cuda::std::min(static_cast<size_t>(str.size_bytes()), sizeof(uint32_t));
    memcpy(&data, str.data(), size);
    return __byte_perm(data, 0, 0x0123);  // unswaps bytes for sorting
  }
};

/**
 * @brief Builds a prefix string from a chars index
 */
struct index_to_prefix_fn {
  cudf::device_span<char const> chars;
  __device__ uint32_t operator()(cudf::size_type idx) const
  {
    auto const size = cuda::std::min((chars.size() - idx), sizeof(uint32_t));
    uint32_t data   = 0;
    memcpy(&data, chars.data() + idx, size);
    return __byte_perm(data, 0, 0x0123);  // unswaps bytes for sorting
  }
};

std::unique_ptr<cudf::column> resolve_duplicates_pair_impl(
  cudf::strings_column_view const& input1,
  cudf::device_span<cudf::size_type const> indices1,
  cudf::strings_column_view const& input2,
  cudf::device_span<cudf::size_type const> indices2,
  cudf::size_type min_width,
  rmm::cuda_stream_view stream,
  rmm::device_async_resource_ref mr)
{
  CUDF_EXPECTS(min_width > 8, "min_width should be at least 8", std::invalid_argument);
  auto d_strings1 = cudf::column_device_view::create(input1.parent(), stream);
  auto d_strings2 = cudf::column_device_view::create(input2.parent(), stream);

  auto [first_offset1, last_offset1] =
    cudf::strings::detail::get_first_and_last_offset(input1, stream);
  auto [first_offset2, last_offset2] =
    cudf::strings::detail::get_first_and_last_offset(input2, stream);

  auto d_input_chars1 = input1.chars_begin(stream) + first_offset1;
  auto d_input_chars2 = input2.chars_begin(stream) + first_offset2;
  auto chars_size1    = last_offset1 - first_offset1;
  auto chars_size2    = last_offset2 - first_offset2;
  CUDF_EXPECTS(
    min_width < chars_size1, "min_width value cannot exceed the input size", std::invalid_argument);
  CUDF_EXPECTS(
    min_width < chars_size2, "min_width value cannot exceed the input size", std::invalid_argument);

  auto const chars_span1 = cudf::device_span<char const>(d_input_chars1, chars_size1);
  auto const chars_span2 = cudf::device_span<char const>(d_input_chars2, chars_size2);

  auto const itr1 =
    thrust::make_transform_iterator(indices1.begin(), index_to_prefix_fn{chars_span1});
  auto const end1 = itr1 + indices1.size();
  auto const itr2 =
    thrust::make_transform_iterator(indices2.begin(), index_to_string_fn{chars_span2});
  auto const end2 = itr2 + indices2.size();

  // vectorized lower-bound and upper-bound of prefix strings improves performance of the
  // transform(find_duplicates_fn) by 2x
  auto prefixes = rmm::device_uvector<uint32_t>(indices2.size(), stream);  // 4x input2
  thrust::transform(
    rmm::exec_policy_nosync(stream), itr2, end2, prefixes.begin(), string_to_prefix_fn{});
  auto lb_ids = rmm::device_uvector<cudf::size_type>(indices1.size(), stream);  // 4x input1
  thrust::lower_bound(
    rmm::exec_policy_nosync(stream), prefixes.begin(), prefixes.end(), itr1, end1, lb_ids.begin());
  auto ub_ids = rmm::device_uvector<cudf::size_type>(indices1.size(), stream);  // 4x input1
  thrust::upper_bound(
    rmm::exec_policy_nosync(stream), prefixes.begin(), prefixes.end(), itr1, end1, ub_ids.begin());

  // resolve duplicates by searching for input2 with strings from input1
  auto fd_fn =
    find_duplicates_fn{chars_span1, chars_span2, min_width, indices1, indices2, lb_ids, ub_ids};
  auto sizes = rmm::device_uvector<int16_t>(indices1.size(), stream);  // 2x input1
  thrust::transform(rmm::exec_policy_nosync(stream),
                    thrust::counting_iterator<cudf::size_type>(0),
                    thrust::counting_iterator<cudf::size_type>(sizes.size()),
                    sizes.begin(),
                    fd_fn);

  // candidate duplicates from input1 have matched the input2 data at least min_width;
  // this means any duplicates in both inputs should be reflected in indices1/sizes;
  // so we should be able to filter/collapse the results using only indices1/sizes
  auto const dup_count =
    sizes.size() - thrust::count(rmm::exec_policy(stream), sizes.begin(), sizes.end(), 0);
  auto dup_indices = rmm::device_uvector<cudf::size_type>(dup_count, stream);

  // remove the non-candidate entries from indices and sizes
  thrust::remove_copy_if(
    rmm::exec_policy(stream),
    indices1.begin(),
    indices1.end(),
    thrust::counting_iterator<cudf::size_type>(0),
    dup_indices.begin(),
    [d_sizes = sizes.data()] __device__(cudf::size_type idx) -> bool { return d_sizes[idx] == 0; });
  auto end = thrust::remove(rmm::exec_policy(stream), sizes.begin(), sizes.end(), 0);
  sizes.resize(cuda::std::distance(sizes.begin(), end), stream);

  // sort the resulting indices/sizes for overlap filtering
  thrust::sort_by_key(
    rmm::exec_policy_nosync(stream), dup_indices.begin(), dup_indices.end(), sizes.begin());

  // produce final duplicates for make_strings_column and collapse any overlapping candidates
  auto duplicates =
    rmm::device_uvector<cudf::strings::detail::string_index_pair>(dup_count, stream);
  thrust::transform(rmm::exec_policy_nosync(stream),
                    thrust::counting_iterator<cudf::size_type>(0),
                    thrust::counting_iterator<cudf::size_type>(dup_indices.size()),
                    duplicates.begin(),
                    collapse_overlaps_fn{chars_span1.data(), dup_indices.data(), sizes.data()});

  // filter out the remaining non-viable candidates
  duplicates.resize(
    cuda::std::distance(
      duplicates.begin(),
      thrust::remove(
        rmm::exec_policy(stream), duplicates.begin(), duplicates.end(), string_index{nullptr, 0})),
    stream);

  // sort result by size descending (should be very fast)
  thrust::sort(rmm::exec_policy_nosync(stream),
               duplicates.begin(),
               duplicates.end(),
               [] __device__(auto lhs, auto rhs) -> bool { return lhs.second > rhs.second; });

  // ironically remove duplicates from the sorted list
  duplicates.resize(
    cuda::std::distance(duplicates.begin(),
                        thrust::unique(rmm::exec_policy(stream),
                                       duplicates.begin(),
                                       duplicates.end(),
                                       [] __device__(auto lhs, auto rhs) -> bool {
                                         return cudf::string_view(lhs.first, lhs.second) ==
                                                cudf::string_view(rhs.first, rhs.second);
                                       })),
    stream);

  return cudf::strings::detail::make_strings_column(
    duplicates.begin(), duplicates.end(), stream, mr);
}
}  // namespace

std::unique_ptr<cudf::column> resolve_duplicates_pair(
  cudf::strings_column_view const& input1,
  cudf::device_span<cudf::size_type const> indices1,
  cudf::strings_column_view const& input2,
  cudf::device_span<cudf::size_type const> indices2,
  cudf::size_type min_width,
  rmm::cuda_stream_view stream,
  rmm::device_async_resource_ref mr)
{
  // force the 2nd input to be the smaller one
  return (indices1.size() < indices2.size())
           ? resolve_duplicates_pair_impl(input2, indices2, input1, indices1, min_width, stream, mr)
           : resolve_duplicates_pair_impl(
               input1, indices1, input2, indices2, min_width, stream, mr);
}

}  // namespace detail

std::unique_ptr<rmm::device_uvector<cudf::size_type>> build_suffix_array(
  cudf::strings_column_view const& input,
  cudf::size_type min_width,
  rmm::cuda_stream_view stream,
  rmm::device_async_resource_ref mr)
{
  CUDF_FUNC_RANGE();
  return detail::build_suffix_array(input, min_width, stream, mr);
}

std::unique_ptr<cudf::column> resolve_duplicates(cudf::strings_column_view const& input,
                                                 cudf::device_span<cudf::size_type const> indices,
                                                 cudf::size_type min_width,
                                                 rmm::cuda_stream_view stream,
                                                 rmm::device_async_resource_ref mr)
{
  CUDF_FUNC_RANGE();
  return detail::resolve_duplicates(input, indices, min_width, stream, mr);
}

std::unique_ptr<cudf::column> resolve_duplicates_pair(
  cudf::strings_column_view const& input1,
  cudf::device_span<cudf::size_type const> indices1,
  cudf::strings_column_view const& input2,
  cudf::device_span<cudf::size_type const> indices2,
  cudf::size_type min_width,
  rmm::cuda_stream_view stream,
  rmm::device_async_resource_ref mr)
{
  CUDF_FUNC_RANGE();
  return detail::resolve_duplicates_pair(input1, indices1, input2, indices2, min_width, stream, mr);
}

}  // namespace nvtext
