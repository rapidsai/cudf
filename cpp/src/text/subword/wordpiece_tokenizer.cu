#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2020-2021, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <text/subword/detail/hash_utils.cuh>
#include <text/subword/detail/tokenizer_utils.cuh>
#include <text/subword/detail/wordpiece_tokenizer.hpp>

#include <cudf/detail/utilities/cuda.cuh>
#include <cudf/strings/string_view.cuh>
#include <cudf/utilities/error.hpp>
#include <nvtext/subword_tokenize.hpp>

#include <rmm/cuda_stream_view.hpp>
#include <rmm/exec_policy.hpp>

#include <thrust/for_each.h>
#include <thrust/remove.h>
#include <thrust/transform_scan.h>

namespace nvtext {
namespace detail {
namespace {
/**
 * @brief Initializes the token-ids, word-indices, and token counts vectors.
 *
 * Each thread process a single code point from `code_points`.
 * This also locates the start and end of each word within the `code_points` buffer.
 * A word start is identified as a non-space character that appears right after a space.
 * A word end is identified as a space character that appears right after a non-space one.
 * If the code point at this thread does not represent a word start or word end,
 * a max uint32_t value is written to the appropriate vector instead.
 * A post processing step is required to filter the relevant values in these
 * vectors.
 *
 * It is guaranteed that the same number of valid values will be written to both the
 * start and end indices and that after the select step, the two arrays will be aligned.
 * That is, `start_word_indices[word]` and `end_word_indices[word]` are the start and
 * end for the same word.
 *
 * Memory required is 13 bytes per code point values:
 * - 4 bytes each for `start_word_indices` and `end_word_indices`
 * - 4 bytes for each `token_ids`
 * - 1 byte for each each `tokens_per_word`
 * Also, there is a code point value for each byte in the input strings.
 *
 * @param code_points[in] A pointer to the code points in the strings after normalization.
 * @param start_word_indices[out] An array of size `num_code_points` which will contain the
 *        starting index for each word.
 * @param end_word_indices[out] An array of size `num_code_points` which will contain the
 *        ending index for each word.
 * @param num_code_points The total number of code_points.
 * @param token_ids[out] An array of size `num_code_points` which will hold the token ids.
 *        This kernel just sets all the values to max uint32_t.
 * @param tokens_per_word[out] An array of size `num_code_points` which hold the number of
 *        tokens. This kernel just sets all the values to 0.
 */
__global__ void init_data_and_mark_word_start_and_ends(uint32_t const* code_points,
                                                       uint32_t* start_word_indices,
                                                       uint32_t* end_word_indices,
                                                       size_t num_code_points,
                                                       uint32_t* token_ids,
                                                       uint8_t* tokens_per_word)
{
  uint32_t char_for_thread = blockDim.x * blockIdx.x + threadIdx.x;

  // Deal with the start_word_indices array
  if (char_for_thread < num_code_points) {
    uint32_t val_to_write = std::numeric_limits<uint32_t>::max();
    if ((code_points[char_for_thread] != SPACE_CODE_POINT) && (char_for_thread > 0) &&
        (code_points[char_for_thread - 1] == SPACE_CODE_POINT)) {
      val_to_write = char_for_thread;
    }
    start_word_indices[char_for_thread] = val_to_write;

    // Deal with the end_word_indices_array
    val_to_write = std::numeric_limits<uint32_t>::max();
    if ((code_points[char_for_thread] != SPACE_CODE_POINT) &&
        (char_for_thread + 1 < num_code_points) &&
        (code_points[char_for_thread + 1] == SPACE_CODE_POINT)) {
      val_to_write = char_for_thread + 1;
    }
    end_word_indices[char_for_thread] = val_to_write;

    token_ids[char_for_thread]       = std::numeric_limits<uint32_t>::max();
    tokens_per_word[char_for_thread] = 0;
  }
}

/**
 * @brief Resolves the string boundaries for the start and end words.
 *
 * This kernel should be called after `init_data_and_mark_word_start_and_ends` with at
 * least `num_strings` total threads.
 *
 * The start and end indices are updated to honor the string boundaries
 * within the strings array. This corrects any word ranges that span across
 * individual strings.
 *
 * @param code_points A pointer to the code points in the strings.
 * @param strings_offsets An array containing the index of the starting character of each string
 *        with an extra space at the end containing the total number of characters. As a result,
 *        this array is of length num_strings + 1.
 * @param start_word_indices An array which will contain the starting index for each word scattered
 *        throughout. If an index does not represent a word start, the max-uint32_t value is written
 *        to indicate this.
 * @param end_word_indices An array which will contain the one past the end index for each word
 *        scattered throughout. If an index does not represent a word end, the max uint32_t value is
 *        written to indicate this.
 * @param num_strings The total number of strings to be processed.
 */
__global__ void mark_string_start_and_ends(uint32_t const* code_points,
                                           uint32_t const* strings_offsets,
                                           uint32_t* start_word_indices,
                                           uint32_t* end_word_indices,
                                           uint32_t num_strings)
{
  uint32_t idx = blockDim.x * blockIdx.x + threadIdx.x;
  // Ensure the starting character of each strings is written to the word start array.
  if (idx <= num_strings) {
    auto const offset = strings_offsets[idx];

    if ((idx < num_strings) && (code_points[offset] != SPACE_CODE_POINT)) {
      start_word_indices[offset] = offset;
    }

    if ((offset > 0) && (code_points[offset - 1] != SPACE_CODE_POINT)) {
      end_word_indices[offset - 1] = offset;
    }
  }
}

/**
 * @brief Currently supported special tokens.
 *
 * Code logic expects these to be 3 upper-case characters along
 * with a single trailing space.
 */
__constant__ char special_tokens[35]{"BOS EOS UNK SEP PAD CLS MASK "};
constexpr cudf::size_type MIN_ST_WIDTH = 4;  // Min token size in special_tokens
constexpr cudf::size_type MAX_ST_WIDTH = 5;  // Max token size in special_tokens

struct mark_special_tokens {
  /**
   * @brief Check given code-point array to the list of known
   * special tokens.
   */
  __device__ bool is_special_token(uint32_t const* token, cudf::size_type size) const
  {
    if (size < MIN_ST_WIDTH || size > MAX_ST_WIDTH) return false;
    char str_token[MAX_ST_WIDTH];
    // convert code-points to chars
    thrust::transform(thrust::seq, token, token + size, str_token, [](uint32_t cp) {
      // also upper-case them to match again special_tokens array
      return static_cast<char>(cp >= 'a' ? cp - 'a' + 'A' : cp);
    });
    // search the special tokens array for the str_token
    cudf::string_view tokens(special_tokens, sizeof(special_tokens));
    return tokens.find(str_token, size) >= 0;
  }

  /**
   * @brief Check code-points for special tokens and adjust indices.
   *
   * Tokens will appear in the `code_points` array as:
   * `_[_ttt_]_` where `_` are single space characters and
   *                   ttt is the variable-length token name
   *
   * The logic below uses the following variables to represent position
   * values in the `code_points` array after locating a special token:
   * ```
   * _ [ _ t t t _  ] _
   *   ^   ^     ^  ^
   *   si  sp    ep ei
   * ```
   * where `si` is `start_index`
   *       `sp` is `start_pos`
   *       `ep` is `end_pos`
   *       `ei` is `end_index`
   *
   * When a special token is found, the `code_points` are adjusted
   * to remove the spaces and capitalize the name.
   * ```
   * _ [ _ t t t _ ] _  is updated to
   * _ [ T T T ] _ ] _
   * ```
   * This is required for the downstream word-piece tokenizer to
   * match it to the vocabulary hash table.
   *
   * The `start_word_indices` and `end_word_indices` are updated to
   * identify the token and to ignore the extra trailing `]` character.
   */
  __device__ void operator()(size_t idx) const
  {
    uint32_t const start_index = start_word_indices[idx];
    if ((start_index == std::numeric_limits<uint32_t>::max()) ||
        ((start_index + MIN_ST_WIDTH + 2) > num_code_points))
      return;
    if (code_points[start_index] != '[') return;

    // check for matching end bracket
    uint32_t const start_pos = start_index + 2;  // after the space delimiter
    // search for next start-word and then check it is a ']'
    uint32_t const end_index = [&] {
      auto const begin = start_word_indices + start_pos;
      auto const width =
        std::min(static_cast<size_t>(MAX_ST_WIDTH + 1), (num_code_points - start_pos));
      auto const end = begin + width;
      // checking the next start-word is more reliable than arbitrarily searching for ']'
      // in case the text is split across string rows
      auto const iter = thrust::find_if(thrust::seq, begin + 1, end, [](auto swi) {
        return swi != std::numeric_limits<uint32_t>::max();
      });
      return iter == end ? start_index : static_cast<uint32_t>(iter - start_word_indices);
    }();
    if (code_points[end_index] != ']') return;

    // check for special token
    auto const size = static_cast<cudf::size_type>(end_index - start_pos);
    if (!is_special_token(code_points + start_pos, size)) return;

    // special token found
    // adjust code-points
    auto const end_pos = end_index - 2;
    // change _[_ttt_]_ to _[TTT]_
    for (auto left_idx = start_pos - 1; left_idx <= end_pos; ++left_idx) {
      auto const cp         = code_points[left_idx + 1];
      code_points[left_idx] = cp >= 'a' ? cp - 'a' + 'A' : cp;
    }
    code_points[end_pos] = ']';

    // erase the intermediate indices
    thrust::fill(thrust::seq,
                 start_word_indices + start_index + 1,  // keep the first one
                 start_word_indices + end_index + 1,
                 std::numeric_limits<uint32_t>::max());
    thrust::fill(thrust::seq,
                 end_word_indices + start_index,
                 end_word_indices + end_index + 1,
                 std::numeric_limits<uint32_t>::max());

    // reset the new end-word index
    end_word_indices[end_pos] = end_pos + 1;
  }

  uint32_t* const code_points;
  uint32_t* const start_word_indices;
  uint32_t* const end_word_indices;
  size_t const num_code_points;
};

/**
 * @brief Converts words into token ids.
 *
 * Each thread is assigned a word to convert based on the `hash_table`. Each thread converts
 * its word and writes the number of tokens it found in the `tokens_per_word` array.
 *
 * The `tokens_per_word` array is kept to the length `num_code_points + 1`. This means each thread
 * can write its number of tokens to the `tokens_per_word` corresponding to the starting
 * character of each word. Since strings must start at some word, we can prefix sum this array
 * and use the strings_lengths code point offsets to directly index the number of tokens in each
 * string.
 *
 * The `token_ids` array should be initialized to the max uint32_t before calling this kernel.
 *
 * @param code_points An array containing all of the code points to be processed
 * @param hash_table An array containing the flattened hash table with key, value pairs
 *        packed in 64-bits
 * @param bin_coefficients A pointer to the GPU pointer containing the hashing parameters for
 *        each hash bin on the GPU.
 * @param bin_offsets: A pointer to the GPU pointer containing the start index of each bin in
 *        the flattened hash table.
 * @param token_ids The index for each token found during tokenization. This is of length
 *        num_code_points. In most cases, multiple characters will collapse to one token. In these
 *        cases, the max uint32_t will be in place. Cub will be used later to filter out these
 *        invalid ids later.
 * @param word_starts An array of length `num_code_points`. The first total word elements contains
 *        the index of the first character for each word.
 * @param word_ends An array of length num_code_points. The first total_words elements contains the
 *        past the end index for each word. This array is kept aligned with the initial
 *        token_ids array containing the word start code points.
 *        `word_ends[word] - filtered_start_indices[word] = word_length`
 * @param tokens_per_word An array of size num_code_points that will contain the number of tokens in
 *        each word in a string. This array can be exclusive summed and the result used in
 *        conjunction with the strings lengths array to find the tokens in each string. This is
 *        possible since the number of tokens in each word will be placed at the index corresponding
 *        to the start character of a word. If we assume prefix_summed is the prefix sum of the
 *        tokens_per_word array, then `prefix_summed[strings_lengths[string_idx] - 1]` is the number
 *        of tokens found before the start of string.
 * @param unk_token_id The token id to be place for unknown tokens
 * @param max_word_length The maximum length of a word. Any word longer than this length is
 *        replaced by the unknown token.
 * @param total_words The total number of white space separated words
 * @param outer_hash_a_param The a parameter for the outer hash
 * @param outer_hash_b_param: The b parameter for the outer hash
 * @param num_outer_bins: The number of bins for the outer hash
 */
__global__ void kernel_wordpiece_tokenizer(uint32_t const* code_points,
                                           uint64_t const* hash_table,
                                           uint64_t const* bin_coefficients,
                                           uint16_t const* bin_offsets,
                                           uint16_t unk_token_id,
                                           uint32_t outer_hash_a_param,
                                           uint32_t outer_hash_b_param,
                                           uint16_t num_outer_bins,
                                           uint32_t const* word_starts,
                                           uint32_t const* word_ends,
                                           uint32_t max_word_length,
                                           uint32_t total_words,
                                           uint32_t* token_ids,
                                           uint8_t* tokens_per_word)
{
  uint32_t const word_to_tokenize = blockDim.x * blockIdx.x + threadIdx.x;

  if (word_to_tokenize >= total_words) return;
  // Each thread gets the start code_point offset for each word and resets the token_id memory to
  // the default value. In a post processing step, all of these values will be removed.
  auto const token_start = word_starts[word_to_tokenize];
  auto const token_end   = word_ends[word_to_tokenize];
  auto const word_length = token_end - token_start;

  // The sdbm hash of "##"
  constexpr uint32_t hashtag_hash = 2296000;
  uint16_t num_values_tokenized   = 0;
  // initialize start, end
  uint32_t start = token_start;
  uint32_t end   = token_end;

  if (word_length > max_word_length) {
    start                        = token_end;
    num_values_tokenized         = 1;
    token_ids[token_start]       = unk_token_id;
    tokens_per_word[token_start] = num_values_tokenized;
  }

  while (start < token_end) {
    end = token_end;
    // init token_id to no token
    int token_id = -1;
    // compute current length
    uint32_t const length = token_end - start;
    uint64_t substr_hash =
      sdbm_hash(code_points + start, length, start == token_start ? 0 : hashtag_hash);
    while (start < end) {
      token_id = retrieve(substr_hash,
                          outer_hash_a_param,
                          outer_hash_b_param,
                          num_outer_bins,
                          hash_table,
                          bin_coefficients,
                          bin_offsets);
      if (token_id != -1) { break; }
      --end;
      // Pop off the last value from the substr hash
      substr_hash = prev_sdbm_hash(substr_hash, code_points[end]);
    }

    if (token_id == -1) {
      end      = token_end;
      token_id = unk_token_id;
      // We need to clean up the global array. This case is very uncommon.
      //  Only 0.016% of words cannot be resolved to a token from the squad dev set.
      for (uint32_t i = 1; i < num_values_tokenized; ++i) {
        token_ids[token_start + i] = std::numeric_limits<uint32_t>::max();
      }
      num_values_tokenized = 0;
    }

    token_ids[token_start + num_values_tokenized] = token_id;
    ++num_values_tokenized;
    start = end;
  }

  tokens_per_word[token_start] = num_values_tokenized;
}

}  // namespace

wordpiece_tokenizer::wordpiece_tokenizer(hashed_vocabulary const& vocab_table,
                                         uint32_t max_rows_final_tensor,
                                         uint32_t max_sequence_length,
                                         uint32_t stride,
                                         bool do_truncate,
                                         bool do_lower_case,
                                         rmm::cuda_stream_view stream,
                                         uint32_t max_word_length)
  : vocab_table(vocab_table),
    normalizer(stream, do_lower_case),
    max_sequence_length{max_sequence_length},
    stride(stride),
    do_truncate(do_truncate),
    max_word_length{max_word_length}
{
}

uvector_pair wordpiece_tokenizer::tokenize(char const* d_strings,
                                           uint32_t const* d_offsets,
                                           uint32_t num_strings,
                                           rmm::cuda_stream_view stream)
{
  auto cps_and_offsets = normalizer.normalize(d_strings, d_offsets, num_strings, stream);
  tokenize(cps_and_offsets, stream);
  return uvector_pair(std::move(cps_and_offsets.first), std::move(cps_and_offsets.second));
}

struct copy_if_fn {  // inline lambda not allowed in private or protected member function
  __device__ bool operator()(uint32_t cp) { return cp != std::numeric_limits<uint32_t>::max(); }
};

struct tranform_fn {  // just converting uint8 value to uint32
  __device__ uint32_t operator()(uint8_t count) { return count; }
};

void wordpiece_tokenizer::tokenize(uvector_pair& cps_and_offsets, rmm::cuda_stream_view stream)
{
  uint32_t* device_code_points     = cps_and_offsets.first->data();
  size_t const num_code_points     = cps_and_offsets.first->size();
  uint32_t* device_strings_offsets = cps_and_offsets.second->data();
  uint32_t const num_strings       = cps_and_offsets.second->size() - 1;

  const size_t four_byte_cp_chunks = 1 + (num_code_points - 1) / sizeof(uint32_t);
  const size_t rounded_num_cps     = sizeof(uint32_t) * four_byte_cp_chunks;
  rmm::device_uvector<uint8_t> device_tokens_per_word(rounded_num_cps, stream);
  rmm::device_uvector<uint32_t> device_token_ids(num_code_points, stream);
  rmm::device_uvector<uint32_t> device_word_indices(2 * num_code_points, stream);

  // make device_start_word_indices and device_end_word_indices contiguous
  uint32_t* device_start_word_indices = device_word_indices.data();
  uint32_t* device_end_word_indices   = device_start_word_indices + num_code_points;

  cudf::detail::grid_1d const grid_init{static_cast<cudf::size_type>(num_code_points),
                                        THREADS_PER_BLOCK};
  detail::init_data_and_mark_word_start_and_ends<<<grid_init.num_blocks,
                                                   grid_init.num_threads_per_block,
                                                   0,
                                                   stream.value()>>>(device_code_points,
                                                                     device_start_word_indices,
                                                                     device_end_word_indices,
                                                                     num_code_points,
                                                                     device_token_ids.data(),
                                                                     device_tokens_per_word.data());
  CHECK_CUDA(stream.value());

  cudf::detail::grid_1d const grid_mark{static_cast<cudf::size_type>(num_strings + 1),
                                        THREADS_PER_BLOCK};
  detail::mark_string_start_and_ends<<<grid_mark.num_blocks,
                                       grid_mark.num_threads_per_block,
                                       0,
                                       stream.value()>>>(device_code_points,
                                                         device_strings_offsets,
                                                         device_start_word_indices,
                                                         device_end_word_indices,
                                                         num_strings);
  CHECK_CUDA(stream.value());

  // check for special tokens and adjust indices
  thrust::for_each_n(
    rmm::exec_policy(stream),
    thrust::make_counting_iterator<size_t>(0),
    num_code_points,
    mark_special_tokens{
      device_code_points, device_start_word_indices, device_end_word_indices, num_code_points});

  // Now start_word_indices has the word starts scattered throughout the array. We need to select
  // all values not equal to the max uint32_t and place them at the start of the array. We leverage
  // the fact that the start_word_indices and the end_word indices are contiguous to only launch one
  // device select kernel.
  auto itr_end = thrust::remove(rmm::exec_policy(stream),
                                device_word_indices.begin(),
                                device_word_indices.end(),
                                std::numeric_limits<uint32_t>::max());

  // The number of tokens selected will be double the number of words since we
  // select from both the start and end index arrays.
  uint32_t const num_words = thrust::distance(device_word_indices.begin(), itr_end) / 2;

  // We need to change the end_word_indices pointer after the selection is complete
  device_end_word_indices = device_start_word_indices + num_words;

  cudf::detail::grid_1d const grid{static_cast<cudf::size_type>(num_words), THREADS_PER_BLOCK};
  detail::
    kernel_wordpiece_tokenizer<<<grid.num_blocks, grid.num_threads_per_block, 0, stream.value()>>>(
      device_code_points,
      vocab_table.table->view().data<uint64_t>(),
      vocab_table.bin_coefficients->view().data<uint64_t>(),
      vocab_table.bin_offsets->view().data<uint16_t>(),
      vocab_table.unknown_token_id,
      vocab_table.outer_hash_a,
      vocab_table.outer_hash_b,
      vocab_table.num_bins,
      device_start_word_indices,
      device_end_word_indices,
      max_word_length,
      num_words,
      device_token_ids.data(),
      device_tokens_per_word.data());
  CHECK_CUDA(stream.value());

  // Repurpose the input array for the token ids. In the worst case, each code point ends up being a
  // token so this will always have enough memory to store the contiguous tokens.
  uint32_t* contiguous_token_ids = device_code_points;
  thrust::copy_if(rmm::exec_policy(stream),
                  device_token_ids.begin(),
                  device_token_ids.end(),
                  contiguous_token_ids,
                  copy_if_fn{});

  // Repurpose start word indices since it is the same size and type as the required output.
  uint32_t* token_id_counts = device_start_word_indices;
  thrust::transform_inclusive_scan(rmm::exec_policy(stream),
                                   device_tokens_per_word.data(),
                                   device_tokens_per_word.data() + num_code_points,
                                   token_id_counts,
                                   tranform_fn{},
                                   thrust::plus<uint32_t>());

  // Update the device_strings_offsets using the token_id_counts
  thrust::for_each_n(rmm::exec_policy(stream),
                     thrust::make_counting_iterator<uint32_t>(1),
                     num_strings,
                     update_strings_lengths_fn{token_id_counts, device_strings_offsets});
}

}  // namespace detail
}  // namespace nvtext
