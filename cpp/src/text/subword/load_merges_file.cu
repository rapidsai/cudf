#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2022, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <text/subword/bpe_tokenizer.cuh>

#include <nvtext/bpe_tokenize.hpp>

#include <cudf/column/column_factories.hpp>
#include <cudf/detail/iterator.cuh>
#include <cudf/detail/nvtx/ranges.hpp>
#include <cudf/detail/utilities/hash_functions.cuh>
#include <cudf/detail/utilities/vector_factories.hpp>
#include <cudf/utilities/default_stream.hpp>
#include <cudf/utilities/error.hpp>

#include <rmm/cuda_stream_view.hpp>
#include <rmm/device_uvector.hpp>

#include <thrust/functional.h>

#include <fstream>
#include <iostream>
#include <vector>

namespace nvtext {
namespace detail {

namespace {

struct make_pair_function {
  /**
   * @brief Hash the merge pair entry
   */
  __device__ cuco::pair_type<cudf::hash_value_type, cudf::size_type> operator()(cudf::size_type idx)
  {
    auto const result = _hasher(d_strings.element<cudf::string_view>(idx));
    return cuco::make_pair(result, idx);
  }

  string_hasher_type const _hasher;
  cudf::column_device_view const d_strings;
};

/**
 * @brief Loads a text file of merge-pairs into a strings column.
 *
 * The line position in the file indicates the pair's rank.
 *
 * @code{.pseudo}
 * Format of the file:
 * #version ..
 * a1 a2
 * b1 b2
 * c1 c2
 * ...
 * @endcode
 *
 * @param filename_merges Path to text file containing merge-pairs
 * @return object containing table elements for the BPE function
 */
std::unique_ptr<cudf::column> load_file_to_column(std::string const& filename_merges,
                                                  rmm::cuda_stream_view stream,
                                                  rmm::mr::device_memory_resource* mr)
{
  std::ifstream merges_file(filename_merges);
  CUDF_EXPECTS(merges_file.good(), "Could not open " + filename_merges);

  std::vector<char> chars{};
  std::vector<cudf::offset_type> offsets(1, 0);

  std::string line;
  std::getline(merges_file, line);
  std::string version = "#version";
  if (line.substr(0, version.size()).compare(version) == 0) { std::getline(merges_file, line); }

  // This is a text file delimited only by CR/LF.
  // TODO: Look into using the CSV reader to load the strings column instead.
  while (!line.empty()) {
    chars.insert(chars.end(), std::cbegin(line), std::cend(line));
    offsets.push_back(offsets.back() + line.length());
    std::getline(merges_file, line);
  }

  CUDF_EXPECTS(!chars.empty(), "No data found in " + filename_merges);

  auto d_chars   = cudf::detail::make_device_uvector_async(chars, stream, mr);
  auto d_offsets = cudf::detail::make_device_uvector_async(offsets, stream, mr);
  return cudf::make_strings_column(d_chars, d_offsets);
}

std::unique_ptr<detail::merge_pairs_map_type> initialize_merge_pairs_map(
  cudf::strings_column_view const& input, rmm::cuda_stream_view stream)
{
  // Ensure capacity is at least (size/0.7) as documented here:
  // https://github.com/NVIDIA/cuCollections/blob/6ec8b6dcdeceea07ab4456d32461a05c18864411/include/cuco/static_map.cuh#L179-L182
  auto merge_pairs_map = std::make_unique<merge_pairs_map_type>(
    static_cast<size_t>(input.size() * 2),  // capacity is 2x;
    cuco::sentinel::empty_key{std::numeric_limits<cudf::hash_value_type>::max()},
    cuco::sentinel::empty_value{-1},  // empty value is not used
    hash_table_allocator_type{default_allocator<char>{}, stream},
    stream.value());

  auto d_strings = cudf::column_device_view::create(input.parent(), stream);
  make_pair_function pair_func{string_hasher_type{}, *d_strings};
  auto iter = cudf::detail::make_counting_transform_iterator(0, pair_func);

  merge_pairs_map->insert(iter,
                          iter + input.size(),
                          cuco::detail::MurmurHash3_32<cudf::hash_value_type>{},
                          thrust::equal_to<cudf::hash_value_type>{},
                          stream.value());

  return merge_pairs_map;
}

std::unique_ptr<bpe_merge_pairs::bpe_merge_pairs_impl> create_bpe_merge_pairs_impl(
  std::unique_ptr<cudf::column>&& input, rmm::cuda_stream_view stream)
{
  auto merge_pairs = initialize_merge_pairs_map(cudf::strings_column_view(input->view()), stream);
  return std::make_unique<nvtext::bpe_merge_pairs::bpe_merge_pairs_impl>(std::move(input),
                                                                         std::move(merge_pairs));
}

std::unique_ptr<bpe_merge_pairs::bpe_merge_pairs_impl> create_bpe_merge_pairs_impl(
  cudf::strings_column_view const& input,
  rmm::cuda_stream_view stream,
  rmm::mr::device_memory_resource* mr)
{
  return create_bpe_merge_pairs_impl(std::make_unique<cudf::column>(input.parent(), stream, mr),
                                     stream);
}

}  // namespace

std::unique_ptr<bpe_merge_pairs> load_merge_pairs_file(std::string const& filename_merges,
                                                       rmm::cuda_stream_view stream,
                                                       rmm::mr::device_memory_resource* mr)
{
  auto input_column = load_file_to_column(filename_merges, stream, mr);
  return std::make_unique<bpe_merge_pairs>(std::move(input_column), stream, mr);
}

}  // namespace detail

std::unique_ptr<bpe_merge_pairs> load_merge_pairs_file(std::string const& filename_merges,
                                                       rmm::mr::device_memory_resource* mr)
{
  CUDF_FUNC_RANGE();
  return detail::load_merge_pairs_file(filename_merges, cudf::get_default_stream(), mr);
}

bpe_merge_pairs::bpe_merge_pairs_impl::bpe_merge_pairs_impl(
  std::unique_ptr<cudf::column>&& merge_pairs,
  std::unique_ptr<detail::merge_pairs_map_type>&& merge_pairs_map)
  : merge_pairs(std::move(merge_pairs)), merge_pairs_map(std::move(merge_pairs_map))
{
}

bpe_merge_pairs::bpe_merge_pairs(std::unique_ptr<cudf::column>&& input,
                                 rmm::cuda_stream_view stream,
                                 rmm::mr::device_memory_resource*)
  : impl(detail::create_bpe_merge_pairs_impl(std::move(input), stream))
{
}

bpe_merge_pairs::bpe_merge_pairs(cudf::strings_column_view const& input,
                                 rmm::cuda_stream_view stream,
                                 rmm::mr::device_memory_resource* mr)
  : impl(detail::create_bpe_merge_pairs_impl(input, stream, mr))
{
}

bpe_merge_pairs::~bpe_merge_pairs() = default;

cudf::size_type bpe_merge_pairs::get_size() { return impl->merge_pairs->size(); }
std::size_t bpe_merge_pairs::get_map_size() { return impl->merge_pairs_map->get_size(); }

}  // namespace nvtext
