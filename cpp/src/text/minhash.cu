#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2023-2024, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <cudf/column/column.hpp>
#include <cudf/column/column_device_view.cuh>
#include <cudf/column/column_factories.hpp>
#include <cudf/detail/copy.hpp>
#include <cudf/detail/null_mask.hpp>
#include <cudf/detail/nvtx/ranges.hpp>
#include <cudf/detail/sequence.hpp>
#include <cudf/detail/utilities/cuda.cuh>
#include <cudf/hashing/detail/hashing.hpp>
#include <cudf/hashing/detail/murmurhash3_x64_128.cuh>
#include <cudf/hashing/detail/murmurhash3_x86_32.cuh>
#include <cudf/strings/string_view.cuh>
#include <cudf/utilities/default_stream.hpp>
#include <cudf/utilities/error.hpp>
#include <cudf/utilities/memory_resource.hpp>

#include <nvtext/minhash.hpp>

#include <rmm/cuda_stream_view.hpp>
#include <rmm/exec_policy.hpp>

#include <cuda/atomic>
#include <thrust/execution_policy.h>
#include <thrust/fill.h>

#include <limits>

namespace nvtext {
namespace detail {
namespace {

/**
 * @brief Compute the minhash of each string for each seed
 *
 * This is a warp-per-string algorithm where parallel threads within a warp
 * work on substrings of a single string row.
 *
 * @tparam HashFunction hash function to use on each substring
 *
 * @param d_strings Strings column to process
 * @param seeds Seeds for hashing each string
 * @param width Substring window size in characters
 * @param d_hashes Minhash output values for each string
 */
template <
  typename HashFunction,
  typename hash_value_type = std::
    conditional_t<std::is_same_v<typename HashFunction::result_type, uint32_t>, uint32_t, uint64_t>>
CUDF_KERNEL void minhash_kernel(cudf::column_device_view const d_strings,
                                cudf::device_span<hash_value_type const> seeds,
                                cudf::size_type width,
                                hash_value_type* d_hashes)
{
  auto const idx = static_cast<std::size_t>(threadIdx.x + blockIdx.x * blockDim.x);
  if (idx >= (static_cast<std::size_t>(d_strings.size()) *
              static_cast<std::size_t>(cudf::detail::warp_size))) {
    return;
  }

  auto const str_idx  = static_cast<cudf::size_type>(idx / cudf::detail::warp_size);
  auto const lane_idx = static_cast<cudf::size_type>(idx % cudf::detail::warp_size);

  if (d_strings.is_null(str_idx)) { return; }

  auto const d_str    = d_strings.element<cudf::string_view>(str_idx);
  auto const d_output = d_hashes + (str_idx * seeds.size());

  // initialize hashes output for this string
  if (lane_idx == 0) {
    auto const init = d_str.empty() ? 0 : std::numeric_limits<hash_value_type>::max();
    thrust::fill(thrust::seq, d_output, d_output + seeds.size(), init);
  }
  __syncwarp();

  auto const begin = d_str.data() + lane_idx;
  auto const end   = d_str.data() + d_str.size_bytes();

  // each lane hashes 'width' substrings of d_str
  for (auto itr = begin; itr < end; itr += cudf::detail::warp_size) {
    if (cudf::strings::detail::is_utf8_continuation_char(*itr)) { continue; }
    auto const check_str =  // used for counting 'width' characters
      cudf::string_view(itr, static_cast<cudf::size_type>(thrust::distance(itr, end)));
    auto const [bytes, left] = cudf::strings::detail::bytes_to_character_position(check_str, width);
    if ((itr != d_str.data()) && (left > 0)) { continue; }  // true if past the end of the string

    auto const hash_str = cudf::string_view(itr, bytes);
    // hashing with each seed on the same section of the string is 10x faster than
    // computing the substrings for each seed
    for (std::size_t seed_idx = 0; seed_idx < seeds.size(); ++seed_idx) {
      auto const hasher = HashFunction(seeds[seed_idx]);
      // hash substring and store the min value
      if constexpr (std::is_same_v<hash_value_type, uint32_t>) {
        auto const hvalue = hasher(hash_str);
        cuda::atomic_ref<hash_value_type, cuda::thread_scope_block> ref{*(d_output + seed_idx)};
        ref.fetch_min(hvalue, cuda::std::memory_order_relaxed);
      } else {
        // This code path assumes the use of MurmurHash3_x64_128 which produces 2 uint64 values
        // but only uses the first uint64 value as requested by the LLM team.
        auto const hvalue = thrust::get<0>(hasher(hash_str));
        cuda::atomic_ref<hash_value_type, cuda::thread_scope_block> ref{*(d_output + seed_idx)};
        ref.fetch_min(hvalue, cuda::std::memory_order_relaxed);
      }
    }
  }
}

template <
  typename HashFunction,
  typename hash_value_type = std::
    conditional_t<std::is_same_v<typename HashFunction::result_type, uint32_t>, uint32_t, uint64_t>>
std::unique_ptr<cudf::column> minhash_fn(cudf::strings_column_view const& input,
                                         cudf::device_span<hash_value_type const> seeds,
                                         cudf::size_type width,
                                         rmm::cuda_stream_view stream,
                                         rmm::device_async_resource_ref mr)
{
  CUDF_EXPECTS(!seeds.empty(), "Parameter seeds cannot be empty", std::invalid_argument);
  CUDF_EXPECTS(width >= 2,
               "Parameter width should be an integer value of 2 or greater",
               std::invalid_argument);
  CUDF_EXPECTS((static_cast<std::size_t>(input.size()) * seeds.size()) <
                 static_cast<std::size_t>(std::numeric_limits<cudf::size_type>::max()),
               "The number of seeds times the number of input rows exceeds the column size limit",
               std::overflow_error);

  auto const output_type = cudf::data_type{cudf::type_to_id<hash_value_type>()};
  if (input.is_empty()) { return cudf::make_empty_column(output_type); }

  auto const d_strings = cudf::column_device_view::create(input.parent(), stream);

  auto hashes   = cudf::make_numeric_column(output_type,
                                          input.size() * static_cast<cudf::size_type>(seeds.size()),
                                          cudf::mask_state::UNALLOCATED,
                                          stream,
                                          mr);
  auto d_hashes = hashes->mutable_view().data<hash_value_type>();

  constexpr int block_size = 256;
  cudf::detail::grid_1d grid{input.size() * cudf::detail::warp_size, block_size};
  minhash_kernel<HashFunction><<<grid.num_blocks, grid.num_threads_per_block, 0, stream.value()>>>(
    *d_strings, seeds, width, d_hashes);

  return hashes;
}

std::unique_ptr<cudf::column> build_list_result(cudf::strings_column_view const& input,
                                                std::unique_ptr<cudf::column>&& hashes,
                                                cudf::size_type seeds_size,
                                                rmm::cuda_stream_view stream,
                                                rmm::device_async_resource_ref mr)
{
  // build the offsets for the output lists column
  auto const zero = cudf::numeric_scalar<cudf::size_type>(0, true, stream);
  auto const size = cudf::numeric_scalar<cudf::size_type>(seeds_size, true, stream);
  auto offsets    = cudf::detail::sequence(input.size() + 1, zero, size, stream, mr);
  hashes->set_null_mask(rmm::device_buffer{}, 0);  // children have no nulls

  // build the lists column from the offsets and the hashes
  auto result = make_lists_column(input.size(),
                                  std::move(offsets),
                                  std::move(hashes),
                                  input.null_count(),
                                  cudf::detail::copy_bitmask(input.parent(), stream, mr),
                                  stream,
                                  mr);
  // expect this condition to be very rare
  if (input.null_count() > 0) {
    result = cudf::detail::purge_nonempty_nulls(result->view(), stream, mr);
  }
  return result;
}
}  // namespace

std::unique_ptr<cudf::column> minhash(cudf::strings_column_view const& input,
                                      cudf::numeric_scalar<uint32_t> const& seed,
                                      cudf::size_type width,
                                      rmm::cuda_stream_view stream,
                                      rmm::device_async_resource_ref mr)
{
  using HashFunction = cudf::hashing::detail::MurmurHash3_x86_32<cudf::string_view>;
  auto const seeds   = cudf::device_span<uint32_t const>{seed.data(), 1};
  auto hashes        = detail::minhash_fn<HashFunction>(input, seeds, width, stream, mr);
  hashes->set_null_mask(cudf::detail::copy_bitmask(input.parent(), stream, mr), input.null_count());
  return hashes;
}

std::unique_ptr<cudf::column> minhash(cudf::strings_column_view const& input,
                                      cudf::device_span<uint32_t const> seeds,
                                      cudf::size_type width,
                                      rmm::cuda_stream_view stream,
                                      rmm::device_async_resource_ref mr)
{
  using HashFunction = cudf::hashing::detail::MurmurHash3_x86_32<cudf::string_view>;
  auto hashes        = detail::minhash_fn<HashFunction>(input, seeds, width, stream, mr);
  return build_list_result(input, std::move(hashes), seeds.size(), stream, mr);
}

std::unique_ptr<cudf::column> minhash64(cudf::strings_column_view const& input,
                                        cudf::numeric_scalar<uint64_t> const& seed,
                                        cudf::size_type width,
                                        rmm::cuda_stream_view stream,
                                        rmm::device_async_resource_ref mr)
{
  using HashFunction = cudf::hashing::detail::MurmurHash3_x64_128<cudf::string_view>;
  auto const seeds   = cudf::device_span<uint64_t const>{seed.data(), 1};
  auto hashes        = detail::minhash_fn<HashFunction>(input, seeds, width, stream, mr);
  hashes->set_null_mask(cudf::detail::copy_bitmask(input.parent(), stream, mr), input.null_count());
  return hashes;
}

std::unique_ptr<cudf::column> minhash64(cudf::strings_column_view const& input,
                                        cudf::device_span<uint64_t const> seeds,
                                        cudf::size_type width,
                                        rmm::cuda_stream_view stream,
                                        rmm::device_async_resource_ref mr)
{
  using HashFunction = cudf::hashing::detail::MurmurHash3_x64_128<cudf::string_view>;
  auto hashes        = detail::minhash_fn<HashFunction>(input, seeds, width, stream, mr);
  return build_list_result(input, std::move(hashes), seeds.size(), stream, mr);
}
}  // namespace detail

std::unique_ptr<cudf::column> minhash(cudf::strings_column_view const& input,
                                      cudf::numeric_scalar<uint32_t> seed,
                                      cudf::size_type width,
                                      rmm::cuda_stream_view stream,
                                      rmm::device_async_resource_ref mr)
{
  CUDF_FUNC_RANGE();
  return detail::minhash(input, seed, width, stream, mr);
}

std::unique_ptr<cudf::column> minhash(cudf::strings_column_view const& input,
                                      cudf::device_span<uint32_t const> seeds,
                                      cudf::size_type width,
                                      rmm::cuda_stream_view stream,
                                      rmm::device_async_resource_ref mr)
{
  CUDF_FUNC_RANGE();
  return detail::minhash(input, seeds, width, stream, mr);
}

std::unique_ptr<cudf::column> minhash64(cudf::strings_column_view const& input,
                                        cudf::numeric_scalar<uint64_t> seed,
                                        cudf::size_type width,
                                        rmm::cuda_stream_view stream,
                                        rmm::device_async_resource_ref mr)
{
  CUDF_FUNC_RANGE();
  return detail::minhash64(input, seed, width, stream, mr);
}

std::unique_ptr<cudf::column> minhash64(cudf::strings_column_view const& input,
                                        cudf::device_span<uint64_t const> seeds,
                                        cudf::size_type width,
                                        rmm::cuda_stream_view stream,
                                        rmm::device_async_resource_ref mr)
{
  CUDF_FUNC_RANGE();
  return detail::minhash64(input, seeds, width, stream, mr);
}

}  // namespace nvtext
