#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2023-2024, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <cudf/column/column.hpp>
#include <cudf/column/column_device_view.cuh>
#include <cudf/column/column_factories.hpp>
#include <cudf/detail/copy.hpp>
#include <cudf/detail/device_scalar.hpp>
#include <cudf/detail/null_mask.hpp>
#include <cudf/detail/nvtx/ranges.hpp>
#include <cudf/detail/sequence.hpp>
#include <cudf/detail/utilities/cuda.cuh>
#include <cudf/hashing/detail/hashing.hpp>
#include <cudf/hashing/detail/murmurhash3_x64_128.cuh>
#include <cudf/hashing/detail/murmurhash3_x86_32.cuh>
#include <cudf/lists/list_device_view.cuh>
#include <cudf/lists/lists_column_device_view.cuh>
#include <cudf/strings/string_view.cuh>
#include <cudf/utilities/default_stream.hpp>
#include <cudf/utilities/error.hpp>
#include <cudf/utilities/memory_resource.hpp>

#include <nvtext/minhash.hpp>

#include <rmm/cuda_stream_view.hpp>
#include <rmm/exec_policy.hpp>

#include <hip/hip_cooperative_groups.h>
#include <cuda/atomic>
#include <thrust/binary_search.h>
#include <thrust/execution_policy.h>
#include <thrust/fill.h>
#include <thrust/sequence.h>
#include <thrust/sort.h>

#include <limits>

namespace nvtext {
namespace detail {
namespace {

/**
 * @brief Compute the minhash of each string for each seed
 *
 * This is a warp-per-string algorithm where parallel threads within a warp
 * work on substrings of a single string row.
 *
 * @tparam HashFunction hash function to use on each substring
 *
 * @param d_strings Strings column to process
 * @param seeds Seeds for hashing each string
 * @param width Substring window size in characters
 * @param d_hashes Minhash output values for each string
 */
template <
  typename HashFunction,
  typename hash_value_type = std::
    conditional_t<std::is_same_v<typename HashFunction::result_type, uint32_t>, uint32_t, uint64_t>>
CUDF_KERNEL void minhash_kernel(cudf::column_device_view const d_strings,
                                cudf::device_span<hash_value_type const> seeds,
                                cudf::size_type width,
                                hash_value_type* d_hashes)
{
  auto const idx = static_cast<std::size_t>(threadIdx.x + blockIdx.x * blockDim.x);
  if (idx >= (static_cast<std::size_t>(d_strings.size()) *
              static_cast<std::size_t>(cudf::detail::warp_size))) {
    return;
  }

  auto const str_idx  = static_cast<cudf::size_type>(idx / cudf::detail::warp_size);
  auto const lane_idx = static_cast<cudf::size_type>(idx % cudf::detail::warp_size);

  if (d_strings.is_null(str_idx)) { return; }

  auto const d_str    = d_strings.element<cudf::string_view>(str_idx);
  auto const d_output = d_hashes + (str_idx * seeds.size());

  // initialize hashes output for this string
  if (lane_idx == 0) {
    auto const init = d_str.empty() ? 0 : std::numeric_limits<hash_value_type>::max();
    thrust::fill(thrust::seq, d_output, d_output + seeds.size(), init);
  }
  __syncwarp();

  auto const begin = d_str.data() + lane_idx;
  auto const end   = d_str.data() + d_str.size_bytes();

  // each lane hashes 'width' substrings of d_str
  for (auto itr = begin; itr < end; itr += cudf::detail::warp_size) {
    if (cudf::strings::detail::is_utf8_continuation_char(*itr)) { continue; }
    auto const check_str =  // used for counting 'width' characters
      cudf::string_view(itr, static_cast<cudf::size_type>(thrust::distance(itr, end)));
    auto const [bytes, left] = cudf::strings::detail::bytes_to_character_position(check_str, width);
    if ((itr != d_str.data()) && (left > 0)) { continue; }  // true if past the end of the string

    auto const hash_str = cudf::string_view(itr, bytes);
    // hashing with each seed on the same section of the string is 10x faster than
    // computing the substrings for each seed
    for (std::size_t seed_idx = 0; seed_idx < seeds.size(); ++seed_idx) {
      auto const hasher = HashFunction(seeds[seed_idx]);
      // hash substring and store the min value
      if constexpr (std::is_same_v<hash_value_type, uint32_t>) {
        auto const hvalue = hasher(hash_str);
        cuda::atomic_ref<hash_value_type, cuda::thread_scope_block> ref{*(d_output + seed_idx)};
        ref.fetch_min(hvalue, cuda::std::memory_order_relaxed);
      } else {
        // This code path assumes the use of MurmurHash3_x64_128 which produces 2 uint64 values
        // but only uses the first uint64 value as requested by the LLM team.
        auto const hvalue = thrust::get<0>(hasher(hash_str));
        cuda::atomic_ref<hash_value_type, cuda::thread_scope_block> ref{*(d_output + seed_idx)};
        ref.fetch_min(hvalue, cuda::std::memory_order_relaxed);
      }
    }
  }
}

template <
  typename HashFunction,
  typename hash_value_type = std::
    conditional_t<std::is_same_v<typename HashFunction::result_type, uint32_t>, uint32_t, uint64_t>>
std::unique_ptr<cudf::column> minhash_fn(cudf::strings_column_view const& input,
                                         cudf::device_span<hash_value_type const> seeds,
                                         cudf::size_type width,
                                         rmm::cuda_stream_view stream,
                                         rmm::device_async_resource_ref mr)
{
  CUDF_EXPECTS(!seeds.empty(), "Parameter seeds cannot be empty", std::invalid_argument);
  CUDF_EXPECTS(width >= 2,
               "Parameter width should be an integer value of 2 or greater",
               std::invalid_argument);
  CUDF_EXPECTS((static_cast<std::size_t>(input.size()) * seeds.size()) <
                 static_cast<std::size_t>(std::numeric_limits<cudf::size_type>::max()),
               "The number of seeds times the number of input rows exceeds the column size limit",
               std::overflow_error);

  auto const output_type = cudf::data_type{cudf::type_to_id<hash_value_type>()};
  if (input.is_empty()) { return cudf::make_empty_column(output_type); }

  auto const d_strings = cudf::column_device_view::create(input.parent(), stream);

  auto hashes   = cudf::make_numeric_column(output_type,
                                          input.size() * static_cast<cudf::size_type>(seeds.size()),
                                          cudf::mask_state::UNALLOCATED,
                                          stream,
                                          mr);
  auto d_hashes = hashes->mutable_view().data<hash_value_type>();

  constexpr cudf::thread_index_type block_size = 256;
  cudf::detail::grid_1d grid{
    static_cast<cudf::thread_index_type>(input.size()) * cudf::detail::warp_size, block_size};
  minhash_kernel<HashFunction><<<grid.num_blocks, grid.num_threads_per_block, 0, stream.value()>>>(
    *d_strings, seeds, width, d_hashes);

  return hashes;
}

constexpr cudf::thread_index_type block_size = 256;
// for potentially tuning minhash_seed_kernel independently from block_size
constexpr cudf::thread_index_type tile_size = block_size;

// Number of a/b parameter values to process per thread.
// The intermediate values are stored in shared-memory and therefore limits this count.
// This value was found to be an efficient size for both uint32 and uint64
// hash types based on benchmarks.
constexpr cuda::std::size_t params_per_thread = 16;

// Separate kernels are used to process strings above and below this value (in bytes).
constexpr cudf::size_type wide_string_threshold = 1 << 18;  // 256K
// The number of blocks per string for the above-threshold kernel processing.
constexpr cudf::size_type blocks_per_string = 64;
// The above values were determined using the redpajama and books_sample datasets

/**
 * @brief Hashing kernel launched as a thread per tile-size (block or warp)
 *
 * This kernel computes the hashes for each string using the seed and the specified
 * hash function. The width is used to compute rolling substrings to hash over.
 * The hashes are stored in d_hashes to be used in the minhash_permuted_kernel.
 *
 * This kernel also counts the number of strings above the wide_string_threshold
 * and proactively initializes the output values for those strings.
 *
 * @tparam HashFunction The hash function to use for this kernel
 * @tparam hash_value_type Derived from HashFunction result_type
 *
 * @param d_strings The input strings to hash
 * @param seed The seed used for the hash function
 * @param width Width in characters used for determining substrings to hash
 * @param d_hashes The resulting hash values are stored here
 * @param threshold_count Stores the number of strings above wide_string_threshold
 * @param param_count Number of parameters (used for the proactive initialize)
 * @param d_results Final results vector (used for the proactive initialize)
 */
template <typename HashFunction, typename hash_value_type = typename HashFunction::result_type>
CUDF_KERNEL void minhash_seed_kernel(cudf::column_device_view const d_strings,
                                     hash_value_type seed,
                                     cudf::size_type width,
                                     hash_value_type* d_hashes,
                                     cudf::size_type* threshold_count,
                                     cudf::size_type param_count,
                                     hash_value_type* d_results)
{
  auto const tid     = cudf::detail::grid_1d::global_thread_id();
  auto const str_idx = tid / tile_size;
  if (str_idx >= d_strings.size()) { return; }
  if (d_strings.is_null(str_idx)) { return; }

  // retrieve this string's offset to locate the output position in d_hashes
  auto const offsets = d_strings.child(cudf::strings_column_view::offsets_column_index);
  auto const offsets_itr =
    cudf::detail::input_offsetalator(offsets.head(), offsets.type(), d_strings.offset());
  auto const offset     = offsets_itr[str_idx];
  auto const size_bytes = static_cast<cudf::size_type>(offsets_itr[str_idx + 1] - offset);
  if (size_bytes == 0) { return; }

  auto const d_str    = cudf::string_view(d_strings.head<char>() + offset, size_bytes);
  auto const lane_idx = tid % tile_size;

  // hashes for this string/thread are stored here
  auto seed_hashes = d_hashes + offset - offsets_itr[0] + lane_idx;

  auto const begin  = d_str.data() + lane_idx;
  auto const end    = d_str.data() + d_str.size_bytes();
  auto const hasher = HashFunction(seed);

  for (auto itr = begin; itr < end; itr += tile_size, seed_hashes += tile_size) {
    if (cudf::strings::detail::is_utf8_continuation_char(*itr)) {
      *seed_hashes = 0;
      continue;
    }
    auto const check_str =  // used for counting 'width' characters
      cudf::string_view(itr, static_cast<cudf::size_type>(thrust::distance(itr, end)));
    auto const [bytes, left] = cudf::strings::detail::bytes_to_character_position(check_str, width);
    if ((itr != d_str.data()) && (left > 0)) {
      // true itr+width is past the end of the string
      *seed_hashes = 0;
      continue;
    }

    auto const hash_str = cudf::string_view(itr, bytes);
    hash_value_type hv;
    if constexpr (std::is_same_v<hash_value_type, uint32_t>) {
      hv = hasher(hash_str);
    } else {
      hv = thrust::get<0>(hasher(hash_str));
    }
    // disallowing hash to zero case
    *seed_hashes = cuda::std::max(hv, hash_value_type{1});
  }

  // logic appended here so an extra kernel is not required
  if (size_bytes >= wide_string_threshold) {
    if (lane_idx == 0) {
      // count the number of wide strings
      cuda::atomic_ref<cudf::size_type, cuda::thread_scope_device> ref{*threshold_count};
      ref.fetch_add(1, cuda::std::memory_order_relaxed);
    }
    // initialize the output -- only needed for wider strings
    auto d_output = d_results + (str_idx * param_count);
    for (auto i = lane_idx; i < param_count; i += tile_size) {
      d_output[i] = std::numeric_limits<hash_value_type>::max();
    }
  }
}

/**
 * @brief Permutation calculation kernel
 *
 * This kernel uses the hashes from the minhash_seed_kernel and the parameter_a and
 * parameter_b values to compute the final output results.
 * The output is the number of input rows (N) by the number of parameter values (M).
 * Each output[i] is the calculated result for parameter_a/b[0:M].
 *
 * This kernel is launched with either blocks per strings of 1 for strings
 * below the wide_strings_threshold or blocks per string = blocks_per_strings
 * for strings above wide_strings_threshold.
 *
 * @tparam hash_value_type Derived from HashFunction result_type
 * @tparam blocks_per_string Number of blocks used to process each string
 *
 * @param d_strings The input strings to hash
 * @param indices The indices of the strings in d_strings to process
 * @param parameter_a 1st set of parameters for the calculation result
 * @param parameter_b 2nd set of parameters for the calculation result
 * @param width Used for calculating the number of available hashes in each string
 * @param d_hashes The hash values computed in minhash_seed_kernel
 * @param d_results Final results vector of calculate values
 */
template <typename hash_value_type, int blocks_per_string>
CUDF_KERNEL void minhash_permuted_kernel(cudf::column_device_view const d_strings,
                                         cudf::device_span<cudf::size_type const> indices,
                                         cudf::device_span<hash_value_type const> parameter_a,
                                         cudf::device_span<hash_value_type const> parameter_b,
                                         cudf::size_type width,
                                         hash_value_type const* d_hashes,
                                         hash_value_type* d_results)
{
  auto const tid = cudf::detail::grid_1d::global_thread_id();
  auto const idx = (tid / blocks_per_string) / block_size;
  if (idx >= indices.size()) { return; }
  auto const str_idx = indices[idx];
  if (d_strings.is_null(str_idx)) { return; }

  auto const block      = cooperative_groups::this_thread_block();
  int const section_idx = block.group_index().x % blocks_per_string;

  auto const offsets = d_strings.child(cudf::strings_column_view::offsets_column_index);
  auto const offsets_itr =
    cudf::detail::input_offsetalator(offsets.head(), offsets.type(), d_strings.offset());
  auto const offset     = offsets_itr[str_idx];
  auto const size_bytes = static_cast<cudf::size_type>(offsets_itr[str_idx + 1] - offset);

  // number of items to process in this block;
  // last block also includes any remainder values from the size_bytes/blocks_per_string truncation
  // example:
  //  each section_size for string with size 588090 and blocks_per_string=64 is 9188
  //  except the last section which is 9188 + (588090 % 64) = 9246
  auto const section_size =
    (size_bytes / blocks_per_string) +
    (section_idx < (blocks_per_string - 1) ? 0 : size_bytes % blocks_per_string);
  auto const section_offset = section_idx * (size_bytes / blocks_per_string);

  // hash values for this block/section
  auto const seed_hashes = d_hashes + offset - offsets_itr[0] + section_offset;
  // width used here as a max value since a string's char-count <= byte-count
  auto const hashes_size =
    section_idx < (blocks_per_string - 1)
      ? section_size
      : cuda::std::max(static_cast<cudf::size_type>(size_bytes > 0), section_size - width + 1);

  auto const init     = size_bytes == 0 ? 0 : std::numeric_limits<hash_value_type>::max();
  auto const lane_idx = block.thread_rank();
  auto const d_output = d_results + (str_idx * parameter_a.size());

  auto const begin = seed_hashes + lane_idx;
  auto const end   = seed_hashes + hashes_size;

  // constants used in the permutation calculations
  constexpr uint64_t mersenne_prime  = (1UL << 61) - 1;
  constexpr hash_value_type hash_max = std::numeric_limits<hash_value_type>::max();

  // found to be an efficient shared memory size for both hash types
  __shared__ hash_value_type block_values[block_size * params_per_thread];

  for (std::size_t i = 0; i < parameter_a.size(); i += params_per_thread) {
    // initialize this block's chunk of shared memory
    // each thread handles params_per_thread of values
    auto const chunk_values = block_values + (lane_idx * params_per_thread);
    thrust::uninitialized_fill(thrust::seq, chunk_values, chunk_values + params_per_thread, init);
    block.sync();

    auto const param_count =
      cuda::std::min(static_cast<cuda::std::size_t>(params_per_thread), parameter_a.size() - i);

    // each lane accumulates min hashes in its shared memory
    for (auto itr = begin; itr < end; itr += block_size) {
      auto const hv = *itr;
      // 0 is used as a skip sentinel for UTF-8 and trailing bytes
      if (hv == 0) { continue; }

      for (std::size_t param_idx = i; param_idx < (i + param_count); ++param_idx) {
        // permutation formula used by datatrove
        hash_value_type const v =
          ((hv * parameter_a[param_idx] + parameter_b[param_idx]) % mersenne_prime) & hash_max;
        auto const block_idx    = ((param_idx % params_per_thread) * block_size) + lane_idx;
        block_values[block_idx] = cuda::std::min(v, block_values[block_idx]);
      }
    }
    block.sync();

    // reduce each parameter values vector to a single min value;
    // assumes that the block_size > params_per_thread;
    // each thread reduces a block_size of parameter values (thread per parameter)
    if (lane_idx < param_count) {
      auto const values = block_values + (lane_idx * block_size);
      // cooperative groups does not have a min function and hipcub::BlockReduce was slower
      auto const minv =
        thrust::reduce(thrust::seq, values, values + block_size, init, thrust::minimum{});
      if constexpr (blocks_per_string > 1) {
        // accumulates mins for each block into d_output
        cuda::atomic_ref<hash_value_type, cuda::thread_scope_block> ref{d_output[lane_idx + i]};
        ref.fetch_min(minv, cuda::std::memory_order_relaxed);
      } else {
        d_output[lane_idx + i] = minv;
      }
    }
    block.sync();
  }
}

template <typename HashFunction, typename hash_value_type = typename HashFunction::result_type>
std::unique_ptr<cudf::column> minhash_fn(cudf::strings_column_view const& input,
                                         hash_value_type seed,
                                         cudf::device_span<hash_value_type const> parameter_a,
                                         cudf::device_span<hash_value_type const> parameter_b,
                                         cudf::size_type width,
                                         rmm::cuda_stream_view stream,
                                         rmm::device_async_resource_ref mr)
{
  CUDF_EXPECTS(width >= 2,
               "Parameter width should be an integer value of 2 or greater",
               std::invalid_argument);
  CUDF_EXPECTS(!parameter_a.empty(), "Parameters A and B cannot be empty", std::invalid_argument);
  CUDF_EXPECTS(parameter_a.size() == parameter_b.size(),
               "Parameters A and B should have the same number of elements",
               std::invalid_argument);
  CUDF_EXPECTS(
    (static_cast<std::size_t>(input.size()) * parameter_a.size()) <
      static_cast<std::size_t>(std::numeric_limits<cudf::size_type>::max()),
    "The number of parameters times the number of input rows exceeds the column size limit",
    std::overflow_error);

  auto const output_type = cudf::data_type{cudf::type_to_id<hash_value_type>()};
  if (input.is_empty()) { return cudf::make_empty_column(output_type); }

  auto const d_strings = cudf::column_device_view::create(input.parent(), stream);

  auto results =
    cudf::make_numeric_column(output_type,
                              input.size() * static_cast<cudf::size_type>(parameter_a.size()),
                              cudf::mask_state::UNALLOCATED,
                              stream,
                              mr);
  auto d_results = results->mutable_view().data<hash_value_type>();

  cudf::detail::grid_1d grid{static_cast<cudf::thread_index_type>(input.size()) * block_size,
                             block_size};
  auto const hashes_size = input.chars_size(stream);
  auto d_hashes          = rmm::device_uvector<hash_value_type>(hashes_size, stream);
  auto d_threshold_count = cudf::detail::device_scalar<cudf::size_type>(0, stream);

  minhash_seed_kernel<HashFunction>
    <<<grid.num_blocks, grid.num_threads_per_block, 0, stream.value()>>>(*d_strings,
                                                                         seed,
                                                                         width,
                                                                         d_hashes.data(),
                                                                         d_threshold_count.data(),
                                                                         parameter_a.size(),
                                                                         d_results);
  auto const threshold_count = d_threshold_count.value(stream);

  auto indices = rmm::device_uvector<cudf::size_type>(input.size(), stream);
  thrust::sequence(rmm::exec_policy(stream), indices.begin(), indices.end());
  cudf::size_type threshold_index = threshold_count < input.size() ? input.size() : 0;

  // if we counted a split of above/below threshold then
  // compute partitions based on the size of each string
  if ((threshold_count > 0) && (threshold_count < input.size())) {
    auto sizes = rmm::device_uvector<cudf::size_type>(input.size(), stream);
    thrust::transform(rmm::exec_policy_nosync(stream),
                      thrust::counting_iterator<cudf::size_type>(0),
                      thrust::counting_iterator<cudf::size_type>(input.size()),
                      sizes.data(),
                      cuda::proclaim_return_type<cudf::size_type>(
                        [d_strings = *d_strings] __device__(auto idx) -> cudf::size_type {
                          if (d_strings.is_null(idx)) { return 0; }
                          return d_strings.element<cudf::string_view>(idx).size_bytes();
                        }));
    thrust::sort_by_key(
      rmm::exec_policy_nosync(stream), sizes.begin(), sizes.end(), indices.begin());
    auto const lb = thrust::lower_bound(
      rmm::exec_policy_nosync(stream), sizes.begin(), sizes.end(), wide_string_threshold);
    threshold_index = static_cast<cudf::size_type>(thrust::distance(sizes.begin(), lb));
  }

  // handle the strings below the threshold width
  if (threshold_index > 0) {
    auto d_indices = cudf::device_span<cudf::size_type const>(indices.data(), threshold_index);
    cudf::detail::grid_1d grid{static_cast<cudf::thread_index_type>(d_indices.size()) * block_size,
                               block_size};
    minhash_permuted_kernel<hash_value_type, 1>
      <<<grid.num_blocks, grid.num_threads_per_block, 0, stream.value()>>>(
        *d_strings, d_indices, parameter_a, parameter_b, width, d_hashes.data(), d_results);
  }

  // handle the strings above the threshold width
  if (threshold_index < input.size()) {
    auto const count = static_cast<cudf::thread_index_type>(input.size() - threshold_index);
    auto d_indices =
      cudf::device_span<cudf::size_type const>(indices.data() + threshold_index, count);
    cudf::detail::grid_1d grid{count * block_size * blocks_per_string, block_size};
    minhash_permuted_kernel<hash_value_type, blocks_per_string>
      <<<grid.num_blocks, grid.num_threads_per_block, 0, stream.value()>>>(
        *d_strings, d_indices, parameter_a, parameter_b, width, d_hashes.data(), d_results);
  }

  return results;
}

/**
 * @brief Compute the minhash of each list row of strings for each seed
 *
 * This is a warp-per-row algorithm where parallel threads within a warp
 * work on strings in a single list row.
 *
 * @tparam HashFunction hash function to use on each string
 *
 * @param d_input List of strings to process
 * @param seeds Seeds for hashing each string
 * @param d_hashes Minhash output values (one per row)
 */
template <
  typename HashFunction,
  typename hash_value_type = std::
    conditional_t<std::is_same_v<typename HashFunction::result_type, uint32_t>, uint32_t, uint64_t>>
CUDF_KERNEL void minhash_word_kernel(cudf::detail::lists_column_device_view const d_input,
                                     cudf::device_span<hash_value_type const> seeds,
                                     hash_value_type* d_hashes)
{
  auto const idx     = cudf::detail::grid_1d::global_thread_id();
  auto const row_idx = idx / cudf::detail::warp_size;

  if (row_idx >= d_input.size()) { return; }
  if (d_input.is_null(row_idx)) { return; }

  auto const d_row    = cudf::list_device_view(d_input, row_idx);
  auto const d_output = d_hashes + (row_idx * seeds.size());

  // initialize hashes output for this row
  auto const lane_idx = static_cast<cudf::size_type>(idx % cudf::detail::warp_size);
  if (lane_idx == 0) {
    auto const init = d_row.size() == 0 ? 0 : std::numeric_limits<hash_value_type>::max();
    thrust::fill(thrust::seq, d_output, d_output + seeds.size(), init);
  }
  __syncwarp();

  // each lane hashes a string from the input row
  for (auto str_idx = lane_idx; str_idx < d_row.size(); str_idx += cudf::detail::warp_size) {
    auto const hash_str =
      d_row.is_null(str_idx) ? cudf::string_view{} : d_row.element<cudf::string_view>(str_idx);
    for (std::size_t seed_idx = 0; seed_idx < seeds.size(); ++seed_idx) {
      auto const hasher = HashFunction(seeds[seed_idx]);
      // hash string and store the min value
      hash_value_type hv;
      if constexpr (std::is_same_v<hash_value_type, uint32_t>) {
        hv = hasher(hash_str);
      } else {
        // This code path assumes the use of MurmurHash3_x64_128 which produces 2 uint64 values
        // but only uses the first uint64 value as requested by the LLM team.
        hv = thrust::get<0>(hasher(hash_str));
      }
      cuda::atomic_ref<hash_value_type, cuda::thread_scope_block> ref{*(d_output + seed_idx)};
      ref.fetch_min(hv, cuda::std::memory_order_relaxed);
    }
  }
}

template <
  typename HashFunction,
  typename hash_value_type = std::
    conditional_t<std::is_same_v<typename HashFunction::result_type, uint32_t>, uint32_t, uint64_t>>
std::unique_ptr<cudf::column> word_minhash_fn(cudf::lists_column_view const& input,
                                              cudf::device_span<hash_value_type const> seeds,
                                              rmm::cuda_stream_view stream,
                                              rmm::device_async_resource_ref mr)
{
  CUDF_EXPECTS(!seeds.empty(), "Parameter seeds cannot be empty", std::invalid_argument);
  CUDF_EXPECTS((static_cast<std::size_t>(input.size()) * seeds.size()) <
                 static_cast<std::size_t>(std::numeric_limits<cudf::size_type>::max()),
               "The number of seeds times the number of input rows exceeds the column size limit",
               std::overflow_error);

  auto const output_type = cudf::data_type{cudf::type_to_id<hash_value_type>()};
  if (input.is_empty()) { return cudf::make_empty_column(output_type); }

  auto const d_input = cudf::column_device_view::create(input.parent(), stream);

  auto hashes   = cudf::make_numeric_column(output_type,
                                          input.size() * static_cast<cudf::size_type>(seeds.size()),
                                          cudf::mask_state::UNALLOCATED,
                                          stream,
                                          mr);
  auto d_hashes = hashes->mutable_view().data<hash_value_type>();
  auto lcdv     = cudf::detail::lists_column_device_view(*d_input);

  constexpr cudf::thread_index_type block_size = 256;
  cudf::detail::grid_1d grid{
    static_cast<cudf::thread_index_type>(input.size()) * cudf::detail::warp_size, block_size};
  minhash_word_kernel<HashFunction>
    <<<grid.num_blocks, grid.num_threads_per_block, 0, stream.value()>>>(lcdv, seeds, d_hashes);

  return hashes;
}

std::unique_ptr<cudf::column> build_list_result(cudf::column_view const& input,
                                                std::unique_ptr<cudf::column>&& hashes,
                                                cudf::size_type seeds_size,
                                                rmm::cuda_stream_view stream,
                                                rmm::device_async_resource_ref mr)
{
  // build the offsets for the output lists column
  auto const zero = cudf::numeric_scalar<cudf::size_type>(0, true, stream);
  auto const size = cudf::numeric_scalar<cudf::size_type>(seeds_size, true, stream);
  auto offsets    = cudf::detail::sequence(input.size() + 1, zero, size, stream, mr);
  hashes->set_null_mask(rmm::device_buffer{}, 0);  // children have no nulls

  // build the lists column from the offsets and the hashes
  auto result = make_lists_column(input.size(),
                                  std::move(offsets),
                                  std::move(hashes),
                                  input.null_count(),
                                  cudf::detail::copy_bitmask(input, stream, mr),
                                  stream,
                                  mr);
  // expect this condition to be very rare
  if (input.null_count() > 0) {
    result = cudf::detail::purge_nonempty_nulls(result->view(), stream, mr);
  }
  return result;
}
}  // namespace

std::unique_ptr<cudf::column> minhash(cudf::strings_column_view const& input,
                                      cudf::numeric_scalar<uint32_t> const& seed,
                                      cudf::size_type width,
                                      rmm::cuda_stream_view stream,
                                      rmm::device_async_resource_ref mr)
{
  using HashFunction = cudf::hashing::detail::MurmurHash3_x86_32<cudf::string_view>;
  auto const seeds   = cudf::device_span<uint32_t const>{seed.data(), 1};
  auto hashes        = detail::minhash_fn<HashFunction>(input, seeds, width, stream, mr);
  hashes->set_null_mask(cudf::detail::copy_bitmask(input.parent(), stream, mr), input.null_count());
  return hashes;
}

std::unique_ptr<cudf::column> minhash(cudf::strings_column_view const& input,
                                      cudf::device_span<uint32_t const> seeds,
                                      cudf::size_type width,
                                      rmm::cuda_stream_view stream,
                                      rmm::device_async_resource_ref mr)
{
  using HashFunction = cudf::hashing::detail::MurmurHash3_x86_32<cudf::string_view>;
  auto hashes        = detail::minhash_fn<HashFunction>(input, seeds, width, stream, mr);
  return build_list_result(input.parent(), std::move(hashes), seeds.size(), stream, mr);
}

std::unique_ptr<cudf::column> minhash(cudf::strings_column_view const& input,
                                      uint32_t seed,
                                      cudf::device_span<uint32_t const> parameter_a,
                                      cudf::device_span<uint32_t const> parameter_b,
                                      cudf::size_type width,
                                      rmm::cuda_stream_view stream,
                                      rmm::device_async_resource_ref mr)
{
  using HashFunction = cudf::hashing::detail::MurmurHash3_x86_32<cudf::string_view>;
  auto hashes =
    detail::minhash_fn<HashFunction>(input, seed, parameter_a, parameter_b, width, stream, mr);
  return build_list_result(input.parent(), std::move(hashes), parameter_a.size(), stream, mr);
}

std::unique_ptr<cudf::column> minhash64(cudf::strings_column_view const& input,
                                        cudf::numeric_scalar<uint64_t> const& seed,
                                        cudf::size_type width,
                                        rmm::cuda_stream_view stream,
                                        rmm::device_async_resource_ref mr)
{
  using HashFunction = cudf::hashing::detail::MurmurHash3_x64_128<cudf::string_view>;
  auto const seeds   = cudf::device_span<uint64_t const>{seed.data(), 1};
  auto hashes        = detail::minhash_fn<HashFunction>(input, seeds, width, stream, mr);
  hashes->set_null_mask(cudf::detail::copy_bitmask(input.parent(), stream, mr), input.null_count());
  return hashes;
}

std::unique_ptr<cudf::column> minhash64(cudf::strings_column_view const& input,
                                        cudf::device_span<uint64_t const> seeds,
                                        cudf::size_type width,
                                        rmm::cuda_stream_view stream,
                                        rmm::device_async_resource_ref mr)
{
  using HashFunction = cudf::hashing::detail::MurmurHash3_x64_128<cudf::string_view>;
  auto hashes        = detail::minhash_fn<HashFunction>(input, seeds, width, stream, mr);
  return build_list_result(input.parent(), std::move(hashes), seeds.size(), stream, mr);
}

std::unique_ptr<cudf::column> minhash64(cudf::strings_column_view const& input,
                                        uint64_t seed,
                                        cudf::device_span<uint64_t const> parameter_a,
                                        cudf::device_span<uint64_t const> parameter_b,
                                        cudf::size_type width,
                                        rmm::cuda_stream_view stream,
                                        rmm::device_async_resource_ref mr)
{
  using HashFunction = cudf::hashing::detail::MurmurHash3_x64_128<cudf::string_view>;
  auto hashes =
    detail::minhash_fn<HashFunction>(input, seed, parameter_a, parameter_b, width, stream, mr);
  return build_list_result(input.parent(), std::move(hashes), parameter_a.size(), stream, mr);
}

std::unique_ptr<cudf::column> word_minhash(cudf::lists_column_view const& input,
                                           cudf::device_span<uint32_t const> seeds,
                                           rmm::cuda_stream_view stream,
                                           rmm::device_async_resource_ref mr)
{
  using HashFunction = cudf::hashing::detail::MurmurHash3_x86_32<cudf::string_view>;
  auto hashes        = detail::word_minhash_fn<HashFunction>(input, seeds, stream, mr);
  return build_list_result(input.parent(), std::move(hashes), seeds.size(), stream, mr);
}

std::unique_ptr<cudf::column> word_minhash64(cudf::lists_column_view const& input,
                                             cudf::device_span<uint64_t const> seeds,
                                             rmm::cuda_stream_view stream,
                                             rmm::device_async_resource_ref mr)
{
  using HashFunction = cudf::hashing::detail::MurmurHash3_x64_128<cudf::string_view>;
  auto hashes        = detail::word_minhash_fn<HashFunction>(input, seeds, stream, mr);
  return build_list_result(input.parent(), std::move(hashes), seeds.size(), stream, mr);
}
}  // namespace detail

std::unique_ptr<cudf::column> minhash(cudf::strings_column_view const& input,
                                      cudf::numeric_scalar<uint32_t> seed,
                                      cudf::size_type width,
                                      rmm::cuda_stream_view stream,
                                      rmm::device_async_resource_ref mr)
{
  CUDF_FUNC_RANGE();
  return detail::minhash(input, seed, width, stream, mr);
}

std::unique_ptr<cudf::column> minhash(cudf::strings_column_view const& input,
                                      cudf::device_span<uint32_t const> seeds,
                                      cudf::size_type width,
                                      rmm::cuda_stream_view stream,
                                      rmm::device_async_resource_ref mr)
{
  CUDF_FUNC_RANGE();
  return detail::minhash(input, seeds, width, stream, mr);
}

std::unique_ptr<cudf::column> minhash_permuted(cudf::strings_column_view const& input,
                                               uint32_t seed,
                                               cudf::device_span<uint32_t const> parameter_a,
                                               cudf::device_span<uint32_t const> parameter_b,
                                               cudf::size_type width,
                                               rmm::cuda_stream_view stream,
                                               rmm::device_async_resource_ref mr)
{
  CUDF_FUNC_RANGE();
  return detail::minhash(input, seed, parameter_a, parameter_b, width, stream, mr);
}

std::unique_ptr<cudf::column> minhash64(cudf::strings_column_view const& input,
                                        cudf::numeric_scalar<uint64_t> seed,
                                        cudf::size_type width,
                                        rmm::cuda_stream_view stream,
                                        rmm::device_async_resource_ref mr)
{
  CUDF_FUNC_RANGE();
  return detail::minhash64(input, seed, width, stream, mr);
}

std::unique_ptr<cudf::column> minhash64(cudf::strings_column_view const& input,
                                        cudf::device_span<uint64_t const> seeds,
                                        cudf::size_type width,
                                        rmm::cuda_stream_view stream,
                                        rmm::device_async_resource_ref mr)
{
  CUDF_FUNC_RANGE();
  return detail::minhash64(input, seeds, width, stream, mr);
}

std::unique_ptr<cudf::column> minhash64_permuted(cudf::strings_column_view const& input,
                                                 uint64_t seed,
                                                 cudf::device_span<uint64_t const> parameter_a,
                                                 cudf::device_span<uint64_t const> parameter_b,
                                                 cudf::size_type width,
                                                 rmm::cuda_stream_view stream,
                                                 rmm::device_async_resource_ref mr)
{
  CUDF_FUNC_RANGE();
  return detail::minhash64(input, seed, parameter_a, parameter_b, width, stream, mr);
}

std::unique_ptr<cudf::column> word_minhash(cudf::lists_column_view const& input,
                                           cudf::device_span<uint32_t const> seeds,
                                           rmm::cuda_stream_view stream,
                                           rmm::device_async_resource_ref mr)
{
  CUDF_FUNC_RANGE();
  return detail::word_minhash(input, seeds, stream, mr);
}

std::unique_ptr<cudf::column> word_minhash64(cudf::lists_column_view const& input,
                                             cudf::device_span<uint64_t const> seeds,
                                             rmm::cuda_stream_view stream,
                                             rmm::device_async_resource_ref mr)
{
  CUDF_FUNC_RANGE();
  return detail::word_minhash64(input, seeds, stream, mr);
}
}  // namespace nvtext
