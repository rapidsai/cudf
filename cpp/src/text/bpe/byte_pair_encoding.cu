#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2022-2025, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "text/bpe/byte_pair_encoding.cuh"

#include <cudf/column/column_device_view.cuh>
#include <cudf/column/column_factories.hpp>
#include <cudf/detail/cuco_helpers.hpp>
#include <cudf/detail/null_mask.hpp>
#include <cudf/detail/nvtx/ranges.hpp>
#include <cudf/detail/utilities/algorithm.cuh>
#include <cudf/detail/utilities/cuda.cuh>
#include <cudf/detail/utilities/functional.hpp>
#include <cudf/hashing/detail/hashing.hpp>
#include <cudf/hashing/detail/murmurhash3_x86_32.cuh>
#include <cudf/strings/detail/strings_children.cuh>
#include <cudf/strings/detail/utilities.hpp>
#include <cudf/strings/split/split.hpp>
#include <cudf/strings/string_view.cuh>
#include <cudf/utilities/default_stream.hpp>
#include <cudf/utilities/error.hpp>
#include <cudf/utilities/memory_resource.hpp>

#include <nvtext/byte_pair_encoding.hpp>

#include <rmm/cuda_stream_view.hpp>
#include <rmm/device_uvector.hpp>
#include <rmm/exec_policy.hpp>

#include <cuco/static_map.cuh>
#include <cuda/functional>
#include <thrust/copy.h>
#include <thrust/distance.h>
#include <thrust/execution_policy.h>
#include <thrust/find.h>
#include <thrust/functional.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/iterator/discard_iterator.h>
#include <thrust/merge.h>
#include <thrust/pair.h>
#include <thrust/remove.h>
#include <thrust/unique.h>

#include <cstdint>
#include <type_traits>

namespace nvtext {

/**
 * @brief Access the bpe_merge_pairs impl member
 *
 * This is used by the encoder to access the impl member functions.
 *
 * @param bpe The merge pairs struct
 * @return The impl object with detailed, internal member data
 */
bpe_merge_pairs::bpe_merge_pairs_impl const* get_bpe_merge_pairs_impl(bpe_merge_pairs const& bpe)
{
  return bpe.impl;
}

namespace detail {
namespace {

using string_hasher_type = cudf::hashing::detail::MurmurHash3_x86_32<cudf::string_view>;
using hash_value_type    = string_hasher_type::result_type;
using merge_pair_type    = thrust::pair<cudf::string_view, cudf::string_view>;
using cuco_storage       = cuco::storage<1>;

/**
 * @brief Hasher function used for building and using the cuco static-map
 *
 * This takes advantage of heterogeneous lookup feature in cuco static-map which
 * allows inserting with one type (index) and looking up with a different type (merge_pair_type).
 *
 * The merge-pairs are in adjacent rows so each index will access two rows of string values.
 * The hash of each string is combined for the returned result.
 */
struct bpe_hasher {
  cudf::column_device_view const d_strings;
  string_hasher_type hasher{};
  // used by insert
  __device__ hash_value_type operator()(cudf::size_type index) const
  {
    index *= 2;
    auto const lhs = d_strings.element<cudf::string_view>(index);
    auto const rhs = d_strings.element<cudf::string_view>(index + 1);
    return cudf::hashing::detail::hash_combine(hasher(lhs), hasher(rhs));
  }
  // used by find
  __device__ hash_value_type operator()(merge_pair_type const& mp) const
  {
    return cudf::hashing::detail::hash_combine(hasher(mp.first), hasher(mp.second));
  }
};

/**
 * @brief Equal function used for building and using the cuco static-map
 *
 * This takes advantage of heterogeneous lookup feature in cuco static-map which
 * allows inserting with one type (index) and looking up with a different type (merge_pair_type).
 *
 * The merge-pairs are in adjacent rows so each index will access two rows of string values.
 * All rows from the input merge-pairs are unique.
 */
struct bpe_equal {
  cudf::column_device_view const d_strings;
  // used by insert
  __device__ bool operator()(cudf::size_type lhs, cudf::size_type rhs) const noexcept
  {
    return lhs == rhs;  // all rows are unique
  }
  // used by find
  __device__ bool operator()(merge_pair_type const& lhs, cudf::size_type rhs) const noexcept
  {
    rhs *= 2;
    auto const left  = d_strings.element<cudf::string_view>(rhs);
    auto const right = d_strings.element<cudf::string_view>(rhs + 1);
    return (left == lhs.first) && (right == lhs.second);
  }
};

using bpe_probe_scheme = cuco::linear_probing<1, bpe_hasher>;

using merge_pairs_map_type = cuco::static_map<cudf::size_type,
                                              cudf::size_type,
                                              cuco::extent<std::size_t>,
                                              cuda::thread_scope_device,
                                              bpe_equal,
                                              bpe_probe_scheme,
                                              cudf::detail::cuco_allocator<char>,
                                              cuco_storage>;

/**
 * @brief Hasher function used for building and using the cuco static-map
 *
 * This takes advantage of heterogeneous lookup feature in cuco static-map which
 * allows inserting with one type (index) and looking up with a different type (merge_pair_type).
 *
 * Each component of the merge-pairs (left and right) are stored individually in the map.
 */
struct mp_hasher {
  cudf::column_device_view const d_strings;
  string_hasher_type hasher{};
  // used by insert
  __device__ hash_value_type operator()(cudf::size_type index) const
  {
    auto const d_str = d_strings.element<cudf::string_view>(index);
    return hasher(d_str);
  }
  // used by find
  __device__ hash_value_type operator()(cudf::string_view const& d_str) const
  {
    return hasher(d_str);
  }
};

/**
 * @brief Equal function used for building and using the cuco static-map
 *
 * This takes advantage of heterogeneous lookup feature in cuco static-map which
 * allows inserting with one type (index) and looking up with a different type (string).
 */
struct mp_equal {
  cudf::column_device_view const d_strings;
  // used by insert
  __device__ bool operator()(cudf::size_type lhs, cudf::size_type rhs) const noexcept
  {
    auto const left  = d_strings.element<cudf::string_view>(lhs);
    auto const right = d_strings.element<cudf::string_view>(rhs);
    return left == right;
  }
  // used by find
  __device__ bool operator()(cudf::string_view const& lhs, cudf::size_type rhs) const noexcept
  {
    auto const right = d_strings.element<cudf::string_view>(rhs);
    return lhs == right;
  }
};

using mp_probe_scheme = cuco::linear_probing<1, mp_hasher>;

using mp_table_map_type = cuco::static_map<cudf::size_type,
                                           cudf::size_type,
                                           cuco::extent<std::size_t>,
                                           cuda::thread_scope_device,
                                           mp_equal,
                                           mp_probe_scheme,
                                           cudf::detail::cuco_allocator<char>,
                                           cuco_storage>;

std::unique_ptr<detail::merge_pairs_map_type> initialize_merge_pairs_map(
  cudf::column_device_view const& input, rmm::cuda_stream_view stream)
{
  auto merge_pairs_map = std::make_unique<merge_pairs_map_type>(
    static_cast<size_t>(input.size()),
    cuco::empty_key{-1},
    cuco::empty_value{-1},
    bpe_equal{input},
    bpe_probe_scheme{bpe_hasher{input}},
    cuco::thread_scope_device,
    cuco_storage{},
    cudf::detail::cuco_allocator<char>{rmm::mr::polymorphic_allocator<char>{}, stream},
    stream.value());

  auto iter = cudf::detail::make_counting_transform_iterator(
    0,
    cuda::proclaim_return_type<cuco::pair<cudf::size_type, cudf::size_type>>(
      [] __device__(cudf::size_type idx) { return cuco::make_pair(idx, idx); }));

  merge_pairs_map->insert_async(iter, iter + (input.size() / 2), stream.value());
  std::cout << "initialize_merge_pairs_map=" << (int)hipStreamSynchronize(stream.value())
            << std::endl;

  return merge_pairs_map;
}

std::unique_ptr<detail::mp_table_map_type> initialize_mp_table_map(
  cudf::column_device_view const& input, rmm::cuda_stream_view stream)
{
  auto mp_table_map = std::make_unique<mp_table_map_type>(
    static_cast<size_t>(input.size() * 2),
    cuco::empty_key{-1},
    cuco::empty_value{-1},
    mp_equal{input},
    mp_probe_scheme{mp_hasher{input}},
    cuco::thread_scope_device,
    cuco_storage{},
    cudf::detail::cuco_allocator<char>{rmm::mr::polymorphic_allocator<char>{}, stream},
    stream.value());

  auto iter = cudf::detail::make_counting_transform_iterator(
    0,
    cuda::proclaim_return_type<cuco::pair<cudf::size_type, cudf::size_type>>(
      [] __device__(cudf::size_type idx) { return cuco::make_pair(idx, idx); }));

  mp_table_map->insert_async(iter, iter + input.size(), stream.value());
  std::cout << "initialize_mp_table_map=" << (int)hipStreamSynchronize(stream.value())
            << std::endl;

  return mp_table_map;
}

std::unique_ptr<bpe_merge_pairs::bpe_merge_pairs_impl> create_bpe_merge_pairs_impl(
  std::unique_ptr<cudf::column>&& input, rmm::cuda_stream_view stream)
{
  auto d_input      = cudf::column_device_view::create(input->view(), stream);
  auto merge_pairs  = initialize_merge_pairs_map(*d_input, stream);
  auto mp_table_map = initialize_mp_table_map(*d_input, stream);
  return std::make_unique<nvtext::bpe_merge_pairs::bpe_merge_pairs_impl>(
    std::move(input), std::move(d_input), std::move(merge_pairs), std::move(mp_table_map));
}

std::unique_ptr<bpe_merge_pairs::bpe_merge_pairs_impl> create_bpe_merge_pairs_impl(
  cudf::strings_column_view const& input,
  rmm::cuda_stream_view stream,
  rmm::device_async_resource_ref mr)
{
  auto pairs =
    cudf::strings::split_record(input, cudf::string_scalar(" ", true, stream, mr), 1, stream, mr);
  auto content = pairs->release();
  return create_bpe_merge_pairs_impl(std::move(content.children.back()), stream);
}

}  // namespace

std::unique_ptr<bpe_merge_pairs> load_merge_pairs(cudf::strings_column_view const& merge_pairs,
                                                  rmm::cuda_stream_view stream,
                                                  rmm::device_async_resource_ref mr)
{
  CUDF_EXPECTS(!merge_pairs.is_empty(), "Merge pairs must not be empty");
  CUDF_EXPECTS(!merge_pairs.has_nulls(), "Merge pairs may not contain nulls");
  return std::make_unique<bpe_merge_pairs>(merge_pairs, stream, mr);
}

}  // namespace detail

// since column_device_view::create() returns is a little more than
// std::unique_ptr<column_device_view> this helper simplifies the return type for us
using col_device_view = std::invoke_result_t<decltype(&cudf::column_device_view::create),
                                             cudf::column_view,
                                             rmm::cuda_stream_view>;

struct bpe_merge_pairs::bpe_merge_pairs_impl {
  std::unique_ptr<cudf::column> const merge_pairs;
  col_device_view const d_merge_pairs;
  std::unique_ptr<detail::merge_pairs_map_type> merge_pairs_map;  // for BPE
  std::unique_ptr<detail::mp_table_map_type> mp_table_map;        // for locating unpairables

  bpe_merge_pairs_impl(std::unique_ptr<cudf::column>&& merge_pairs,
                       col_device_view&& d_merge_pairs,
                       std::unique_ptr<detail::merge_pairs_map_type>&& merge_pairs_map,
                       std::unique_ptr<detail::mp_table_map_type>&& mp_table_map);

  auto const get_merge_pairs() const { return *d_merge_pairs; }
  auto get_merge_pairs_ref() const { return merge_pairs_map->ref(cuco::op::find); }
  auto get_mp_table_ref() const { return mp_table_map->ref(cuco::op::find); }
};

std::unique_ptr<bpe_merge_pairs> load_merge_pairs(cudf::strings_column_view const& merge_pairs,
                                                  rmm::cuda_stream_view stream,
                                                  rmm::device_async_resource_ref mr)
{
  CUDF_FUNC_RANGE();
  return detail::load_merge_pairs(merge_pairs, stream, mr);
}

bpe_merge_pairs::bpe_merge_pairs_impl::bpe_merge_pairs_impl(
  std::unique_ptr<cudf::column>&& merge_pairs,
  std::unique_ptr<cudf::column_device_view, std::function<void(cudf::column_device_view*)>>&&
    d_merge_pairs,
  std::unique_ptr<detail::merge_pairs_map_type>&& merge_pairs_map,
  std::unique_ptr<detail::mp_table_map_type>&& mp_table_map)
  : merge_pairs(std::move(merge_pairs)),
    d_merge_pairs(std::move(d_merge_pairs)),
    merge_pairs_map(std::move(merge_pairs_map)),
    mp_table_map(std::move(mp_table_map))
{
}

bpe_merge_pairs::bpe_merge_pairs(std::unique_ptr<cudf::column>&& input,
                                 rmm::cuda_stream_view stream,
                                 rmm::device_async_resource_ref)
  : impl(detail::create_bpe_merge_pairs_impl(std::move(input), stream).release())
{
}

bpe_merge_pairs::bpe_merge_pairs(cudf::strings_column_view const& input,
                                 rmm::cuda_stream_view stream,
                                 rmm::device_async_resource_ref mr)
  : impl(detail::create_bpe_merge_pairs_impl(input, stream, mr).release())
{
}

bpe_merge_pairs::bpe_merge_pairs() = default;
bpe_merge_pairs::~bpe_merge_pairs() { delete impl; }

namespace detail {

namespace {

constexpr int block_size = 512;

/**
 * @brief Produces offsets to unpairable locations in the given chars array
 *
 * Launched as a thread per byte of the chars array.
 * The output is non-zero offsets to locations of unpairable substrings.
 * An unpairable substring does not exist in the given map and so will
 * never be paired. Fortunately, this can be used as an artificial
 * boundary providing increased parallelism in the BPE kernel.
 *
 * @tparam MapRefType The type of the map finder object
 */
template <typename MapRefType>
struct bpe_unpairable_offsets_fn {
  cudf::device_span<char const> d_chars;
  int64_t offset;
  MapRefType const d_map;
  __device__ int64_t operator()(int64_t idx)
  {
    if (!cudf::strings::detail::is_begin_utf8_char(d_chars[idx])) { return 0; }

    auto const itr  = d_chars.data() + idx;
    auto const end  = d_chars.end();
    auto const lhs  = cudf::string_view(itr, cudf::strings::detail::bytes_in_utf8_byte(*itr));
    auto const next = itr + lhs.size_bytes();
    auto output     = 0L;
    if (next < end) {
      auto const rhs = cudf::string_view(next, cudf::strings::detail::bytes_in_utf8_byte(*next));
      // see if both halves exist anywhere in the table, if not these are unpairable
      if (d_map.find(lhs) == d_map.end() && d_map.find(rhs) == d_map.end()) {
        output = idx + lhs.size_bytes() + offset;  // offset for artificial boundary
      }
      printf(
        "%ld: [%c/%d],[%c/%d]=%ld\n", idx, *itr, lhs.size_bytes(), *next, rhs.size_bytes(), output);
    }
    return output;
  }
};

/**
 * @brief Performs byte-pair-encoding
 *
 * Computes the locations where the separator will be inserted in `d_spaces_data`.
 * This is launched as a string per block.
 *
 * The process first initializes all characters to 1 per position in `d_spaces_data`.
 * All pairs are realized and their ranks stored in `d_ranks_data`.
 *
 * Iteratively, the minimum rank is located, the corresponding `d_spaces_data` location
 * is set to 0 resulting in new potential pairs. The process repeats accounting for
 * the rank of the newly formed pairs.
 *
 * Once there are no more rankable pairs, the process finishes and the `d_spaces_data`
 * values identify the location to insert the separator.
 *
 * @tparam MapRefType The type of the map finder object
 * @param d_strings Input data
 * @param d_map For looking up individual string candidates
 * @param d_spaces_data Output the location where separator will be inserted
 * @param d_ranks_data Working memory to hold pair ranks
 * @param d_rerank_data Working memory to hold locations where reranking is required
 */
template <typename MapRefType>
CUDF_KERNEL void bpe_parallel_fn(cudf::column_device_view const d_strings,
                                 char const* d_input_chars,
                                 MapRefType const d_map,
                                 int8_t* d_spaces_data,          // working memory
                                 cudf::size_type* d_ranks_data,  // more working memory
                                 int8_t* d_rerank_data           // and one more working memory
)
{
  // string per block
  auto const str_idx =
    static_cast<cudf::size_type>(cudf::detail::grid_1d::global_thread_id() / block_size);
  auto const lane_idx = static_cast<cudf::size_type>(threadIdx.x);

  auto const d_str  = d_strings.element<cudf::string_view>(str_idx);
  auto const offset = thrust::distance(d_input_chars, d_str.data());

  auto const d_spaces   = d_spaces_data + offset;
  auto const end_spaces = d_spaces + d_str.size_bytes();
  auto const d_ranks    = d_ranks_data + offset;
  auto const end_ranks  = d_ranks + d_str.size_bytes();
  auto const d_rerank   = d_rerank_data + offset;
  auto const end_rerank = d_rerank + d_str.size_bytes();

  auto constexpr max_rank = cuda::std::numeric_limits<cudf::size_type>::max();

  __shared__ cudf::size_type block_min_rank;
  using block_reduce = hipcub::BlockReduce<cudf::size_type, block_size>;
  __shared__ typename block_reduce::TempStorage temp_storage;
  auto const num_valid = block_size < d_str.size_bytes() ? block_size : d_str.size_bytes();

  // init all the re-rank identifiers to zero
  for (auto itr = d_rerank + lane_idx; itr < end_rerank; itr += block_size) {
    *itr = 0;
  }
  // init all ranks to max
  for (auto itr = d_ranks + lane_idx; itr < end_ranks; itr += block_size) {
    *itr = max_rank;
  }
  // init all spaces to 1 as appropriate
  for (auto itr = d_spaces + lane_idx; itr < end_spaces; itr += block_size) {
    auto const index = thrust::distance(d_spaces, itr);
    *itr = static_cast<int8_t>(cudf::strings::detail::is_begin_utf8_char(d_str.data()[index]));
  }
  __syncthreads();

  // for finding the next half of a pair
  auto next_substr = [d_str, d_spaces, end = end_spaces](int8_t* begin) {
    auto const next = thrust::find(thrust::seq, begin + 1, end, 1);
    auto const size = static_cast<cudf::size_type>(thrust::distance(begin, next));
    return cudf::string_view(d_str.data() + thrust::distance(d_spaces, begin), size);
  };
  // for locating adjacent pairs after merging a pair
  auto find_prev = [begin = d_spaces](int8_t* ptr) {
    while (ptr > begin && *ptr == 0) {
      --ptr;
    }
    return ptr;
  };

  auto min_rank = max_rank;

  // store all the initial ranks for each pair
  // every character but the first one will have a initial rank
  //
  // Example:
  // string:   abcdefghij
  // spaces:   1111111111
  // ranks:    *948516327
  for (auto itr = d_spaces + lane_idx; itr < end_spaces; itr += block_size) {
    if (*itr == 0) { continue; }  // skips any UTF-8 continuation bytes
    // resolve pair and lookup its rank
    auto const lhs      = next_substr(itr);  // retrieve lhs of the pair
    auto const next_itr = itr + lhs.size_bytes();
    if (next_itr < end_spaces) {
      auto const rhs = next_substr(next_itr);  // retrieve rhs of the pair
      if (!rhs.empty()) {
        auto rank          = max_rank;
        auto const mp      = merge_pair_type{lhs, rhs};
        auto const map_itr = d_map.find(mp);                     // lookup pair in merges table;
        if (map_itr != d_map.end()) { rank = map_itr->second; }  // found a match;
        d_ranks[thrust::distance(d_spaces, next_itr)] = rank;    // store the rank
        if (rank < min_rank) { min_rank = rank; }
      }
    }
  }
  // compute the min rank across the block
  auto const reduce_rank =
    block_reduce(temp_storage).Reduce(min_rank, cudf::detail::minimum{}, num_valid);
  if (lane_idx == 0) {
    block_min_rank = reduce_rank;
    // printf("%d: min_rank=%d\n", str_idx, block_min_rank);
  }
  __syncthreads();

  // loop through the ranks processing the current minimum until there are no more
  while (block_min_rank < max_rank) {
    // search the d_ranks for matches to block_min_rank
    for (auto itr = d_ranks + lane_idx; itr < end_ranks; itr += block_size) {
      if (*itr == block_min_rank) {
        auto ptr = itr - 1;  // check for adjacent min-rank (edge-case)
        while (ptr > d_ranks && *ptr == max_rank) {
          --ptr;
        }
        // set the output value to 0 at this position (erases separator, merges pair)
        // using example string above, the min-rank is 1 at position 5
        // string: abcdefghij
        // spaces: 1111101111  (set position 5 to 0)
        if (*ptr != block_min_rank) { d_spaces[thrust::distance(d_ranks, itr)] = 0; }
      }
    }
    __syncthreads();

    // identify all the re-rank locations (logic above invalidated adjacent pairs)
    // using example string above, the adjacent pairs have to be re-ranked
    // string: abcdefghij
    // spaces: 1111101111 (pair 'e,f' is now merged)
    // rerank: 0000101000 ('ef' and 'fg' need re-ranking as 'd,ef' and 'ef,g'
    for (auto itr = d_ranks + lane_idx; itr < end_ranks; itr += block_size) {
      auto const index = thrust::distance(d_ranks, itr);
      if (*itr == block_min_rank && d_spaces[index] == 0) {
        // find previous pair mid-point
        auto ptr = find_prev(d_spaces + index - 1);
        if (ptr > d_spaces) { d_rerank[thrust::distance(d_spaces, ptr)] = 1; }
        // find next pair mid-point
        ptr = thrust::find(thrust::seq, d_spaces + index + 1, end_spaces, 1);
        if (ptr < end_spaces) { d_rerank[thrust::distance(d_spaces, ptr)] = 1; }
        *itr = max_rank;  // reset this rank
      }
    }
    __syncthreads();

    // compute the ranks for the newly created pairs
    min_rank = max_rank;  // and record the new minimum along the way
    for (auto itr = d_rerank + lane_idx; itr < end_rerank; itr += block_size) {
      auto const index = thrust::distance(d_rerank, itr);
      auto rank        = d_ranks[index];
      if (*itr) {
        *itr = 0;  // reset re-rank
        // build lhs of pair
        auto const ptr  = find_prev(d_spaces + index - 1);
        auto const size = static_cast<cudf::size_type>(thrust::distance(ptr, d_spaces + index));
        auto const lhs  = cudf::string_view(d_str.data() + thrust::distance(d_spaces, ptr), size);
        auto const rhs  = next_substr(d_spaces + index);  // retrieve rhs of pair
        rank            = max_rank;
        if (!rhs.empty()) {
          auto const mp      = merge_pair_type{lhs, rhs};
          auto const map_itr = d_map.find(mp);                     // lookup rank for this pair;
          if (map_itr != d_map.end()) { rank = map_itr->second; }  // found a match
        }
        d_ranks[index] = rank;  // store new rank
      }
      if (rank < min_rank) { min_rank = rank; }
    }

    // re-compute the minimum rank across the block (since new pairs are created above)
    auto const reduce_rank =
      block_reduce(temp_storage).Reduce(min_rank, cudf::detail::minimum{}, num_valid);
    if (lane_idx == 0) { block_min_rank = reduce_rank; }
    __syncthreads();
  }  // if no min ranks are found we are done, otherwise start again
}

/**
 * @brief Computes the output size of each strings row
 *
 * This launches as a string per block.
 * The non-zero values in `d_spaces_data` for each string is added to
 * the current string size to produce the total output bytes.
 *
 * @param d_strings Input data
 * @param d_spaces_data Output the location where separator will be inserted
 * @param d_sizes Output sizes of each row
 */
CUDF_KERNEL void bpe_finalize(cudf::column_device_view const d_strings,
                              char const* d_input_chars,
                              int8_t* d_spaces_data,    // where separators are inserted
                              cudf::size_type* d_sizes  // output sizes of encoded strings
)
{
  // string per block
  auto const str_idx =
    static_cast<cudf::size_type>(cudf::detail::grid_1d::global_thread_id() / block_size);
  auto const lane_idx = static_cast<cudf::size_type>(threadIdx.x);

  if (d_strings.is_null(str_idx)) {
    d_sizes[str_idx] = 0;
    return;
  }
  auto const d_str = d_strings.element<cudf::string_view>(str_idx);
  if (d_str.empty()) {
    d_sizes[str_idx] = 0;
    return;
  }

  auto const offset = thrust::distance(d_input_chars, d_str.data());

  auto const d_spaces   = d_spaces_data + offset;
  auto const end_spaces = d_spaces + d_str.size_bytes();
  auto const num_valid  = block_size < d_str.size_bytes() ? block_size : d_str.size_bytes();

  using block_reduce = hipcub::BlockReduce<cudf::size_type, block_size>;
  __shared__ typename block_reduce::TempStorage temp_storage;

  // reset the first position -- no separator to be added here
  if (lane_idx == 0) { *d_spaces = 0; }

  // compute the output size for this string by counting the resulting separator positions
  auto bytes = 0;
  for (auto itr = d_spaces + lane_idx; itr < end_spaces; itr += block_size) {
    bytes += (*itr > 0);
  }
  auto const total_bytes = block_reduce(temp_storage).Sum(bytes, num_valid);
  if (lane_idx == 0) { d_sizes[str_idx] = total_bytes + d_str.size_bytes(); }
}

}  // namespace

std::unique_ptr<cudf::column> byte_pair_encoding(cudf::strings_column_view const& input,
                                                 bpe_merge_pairs const& merge_pairs,
                                                 cudf::string_scalar const& separator,
                                                 rmm::cuda_stream_view stream,
                                                 rmm::device_async_resource_ref mr)
{
  if (input.is_empty() || input.chars_size(stream) == 0) {
    return cudf::make_empty_column(cudf::type_id::STRING);
  }

  CUDF_EXPECTS(separator.is_valid(stream), "separator parameter must be valid");
  auto const d_separator = separator.value(stream);
  CUDF_EXPECTS(d_separator.size_bytes() == 1, "for now, separator must be a single-byte character");

  auto const d_strings = cudf::column_device_view::create(input.parent(), stream);

  auto const first_offset  = (input.offset() == 0) ? 0L
                                                   : cudf::strings::detail::get_offset_value(
                                                      input.offsets(), input.offset(), stream);
  auto const last_offset   = (input.offset() == 0 && input.size() == input.offsets().size() - 1)
                               ? static_cast<int64_t>(input.chars_size(stream))
                               : cudf::strings::detail::get_offset_value(
                                 input.offsets(), input.size() + input.offset(), stream);
  auto const chars_size    = last_offset - first_offset;
  auto const d_input_chars = input.chars_begin(stream) + first_offset;

  rmm::device_uvector<int8_t> d_spaces(chars_size, stream);  // identifies non-merged pairs
  // used for various purposes below: unpairable-offsets, pair ranks, separator insert positions
  rmm::device_uvector<int64_t> d_working(chars_size, stream);

  auto const chars_begin = thrust::counting_iterator<int64_t>(0);
  auto const chars_end   = thrust::counting_iterator<int64_t>(chars_size);

  {
    // this kernel locates unpairable sections of strings to create artificial string row
    // boundaries; the boundary values are recorded as offsets in d_up_offsets
    auto const d_up_offsets = d_working.data();  // store unpairable offsets here
    auto const mp_map = get_bpe_merge_pairs_impl(merge_pairs)->get_mp_table_ref();  // lookup table
    auto const d_chars_span = cudf::device_span<char const>(d_input_chars, chars_size);
    auto up_fn = bpe_unpairable_offsets_fn<decltype(mp_map)>{d_chars_span, first_offset, mp_map};
    thrust::transform(rmm::exec_policy_nosync(stream), chars_begin, chars_end, d_up_offsets, up_fn);
    auto const up_end =  // remove all but the unpairable offsets
      thrust::remove(rmm::exec_policy_nosync(stream), d_up_offsets, d_up_offsets + chars_size, 0L);
    auto const unpairables = thrust::distance(d_up_offsets, up_end);  // number of unpairables

    // new string boundaries created by combining unpairable offsets with the existing offsets
    auto tmp_offsets = rmm::device_uvector<int64_t>(unpairables + input.size() + 1, stream);
    auto input_offsets =
      cudf::detail::offsetalator_factory::make_input_iterator(input.offsets(), input.offset());
    thrust::merge(rmm::exec_policy_nosync(stream),
                  input_offsets,
                  input_offsets + input.size() + 1,
                  d_up_offsets,
                  up_end,
                  tmp_offsets.begin());
    // remove any adjacent duplicate offsets (i.e. empty or null rows)
    auto const offsets_end =
      thrust::unique(rmm::exec_policy_nosync(stream), tmp_offsets.begin(), tmp_offsets.end());
    auto const offsets_total =
      static_cast<cudf::size_type>(thrust::distance(tmp_offsets.begin(), offsets_end));
    tmp_offsets.resize(offsets_total, stream);

    // temp column created with the merged offsets and the original chars data
    auto const col_offsets = cudf::column_view(cudf::device_span<int64_t const>(tmp_offsets));
    auto const tmp_size    = offsets_total - 1;
    auto const tmp_input   = cudf::column_view(
      input.parent().type(), tmp_size, input.chars_begin(stream), nullptr, 0, 0, {col_offsets});
    auto const d_tmp_strings = cudf::column_device_view::create(tmp_input, stream);

    // launch the byte-pair-encoding kernel on the temp column
    rmm::device_uvector<int8_t> d_rerank(chars_size, stream);  // more working memory;
    rmm::device_uvector<cudf::size_type> d_ranks(chars_size, stream);
    auto const pair_map = get_bpe_merge_pairs_impl(merge_pairs)->get_merge_pairs_ref();
    bpe_parallel_fn<decltype(pair_map)><<<tmp_size, block_size, 0, stream.value()>>>(
      *d_tmp_strings, d_input_chars, pair_map, d_spaces.data(), d_ranks.data(), d_rerank.data());
  }

  // compute the output sizes
  auto output_sizes = rmm::device_uvector<cudf::size_type>(input.size(), stream);
  bpe_finalize<<<input.size(), block_size, 0, stream.value()>>>(
    *d_strings, d_input_chars, d_spaces.data(), output_sizes.data());

  // convert sizes to offsets in-place
  auto [offsets, bytes] = cudf::strings::detail::make_offsets_child_column(
    output_sizes.begin(), output_sizes.end(), stream, mr);

  // build the output: inserting separators to the input character data
  rmm::device_uvector<char> chars(bytes, stream, mr);
  auto d_chars = chars.data();

  auto const d_inserts     = d_working.data();  // stores the insert positions
  auto offsets_at_non_zero = [d_spaces = d_spaces.data()] __device__(auto idx) {
    return d_spaces[idx] > 0;  // separator to be inserted here
  };
  auto const copy_end =
    cudf::detail::copy_if_safe(chars_begin + 1, chars_end, d_inserts, offsets_at_non_zero, stream);

  // this will insert the single-byte separator into positions specified in d_inserts
  auto const sep_char = thrust::constant_iterator<char>(separator.to_string(stream)[0]);
  thrust::merge_by_key(rmm::exec_policy_nosync(stream),
                       d_inserts,      // where to insert separator byte
                       copy_end,       //
                       chars_begin,    // all indices
                       chars_end,      //
                       sep_char,       // byte to insert
                       d_input_chars,  // original data
                       thrust::make_discard_iterator(),
                       d_chars);  // result

  return cudf::make_strings_column(input.size(),
                                   std::move(offsets),
                                   chars.release(),
                                   input.null_count(),
                                   cudf::detail::copy_bitmask(input.parent(), stream, mr));
}

}  // namespace detail

std::unique_ptr<cudf::column> byte_pair_encoding(cudf::strings_column_view const& input,
                                                 bpe_merge_pairs const& merges_table,
                                                 cudf::string_scalar const& separator,
                                                 rmm::cuda_stream_view stream,
                                                 rmm::device_async_resource_ref mr)
{
  CUDF_FUNC_RANGE();
  return detail::byte_pair_encoding(input, merges_table, separator, stream, mr);
}

}  // namespace nvtext
