/*
 * Copyright (c) 2022-2024, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <text/bpe/byte_pair_encoding.cuh>

#include <nvtext/byte_pair_encoding.hpp>

#include <cudf/column/column_factories.hpp>
#include <cudf/detail/iterator.cuh>
#include <cudf/detail/nvtx/ranges.hpp>
#include <cudf/detail/utilities/vector_factories.hpp>
#include <cudf/strings/split/split.hpp>
#include <cudf/utilities/default_stream.hpp>
#include <cudf/utilities/error.hpp>

#include <rmm/cuda_stream_view.hpp>
#include <rmm/device_uvector.hpp>

#include <fstream>
#include <iostream>
#include <vector>

#include <cuda/functional>

namespace nvtext {
namespace detail {
namespace {

std::unique_ptr<detail::merge_pairs_map_type> initialize_merge_pairs_map(
  cudf::column_device_view const& input, rmm::cuda_stream_view stream)
{
  auto merge_pairs_map =
    std::make_unique<merge_pairs_map_type>(static_cast<size_t>(input.size()),
                                           cuco::empty_key{-1},
                                           cuco::empty_value{-1},
                                           bpe_equal{input},
                                           bpe_probe_scheme{bpe_hasher{input}},
                                           cudf::detail::cuco_allocator{stream},
                                           stream.value());

  auto iter = cudf::detail::make_counting_transform_iterator(
    0,
    cuda::proclaim_return_type<cuco::pair<cudf::size_type, cudf::size_type>>(
      [] __device__(cudf::size_type idx) { return cuco::make_pair(idx, idx); }));

  merge_pairs_map->insert_async(iter, iter + (input.size() / 2), stream.value());

  return merge_pairs_map;
}

std::unique_ptr<detail::mp_table_map_type> initialize_mp_table_map(
  cudf::column_device_view const& input, rmm::cuda_stream_view stream)
{
  auto mp_table_map = std::make_unique<mp_table_map_type>(static_cast<size_t>(input.size()),
                                                          cuco::empty_key{-1},
                                                          cuco::empty_value{-1},
                                                          mp_equal{input},
                                                          mp_probe_scheme{mp_hasher{input}},
                                                          cudf::detail::cuco_allocator{stream},
                                                          stream.value());

  auto iter = cudf::detail::make_counting_transform_iterator(
    0,
    cuda::proclaim_return_type<cuco::pair<cudf::size_type, cudf::size_type>>(
      [] __device__(cudf::size_type idx) { return cuco::make_pair(idx, idx); }));

  mp_table_map->insert_async(iter, iter + input.size(), stream.value());

  return mp_table_map;
}

std::unique_ptr<bpe_merge_pairs::bpe_merge_pairs_impl> create_bpe_merge_pairs_impl(
  std::unique_ptr<cudf::column>&& input, rmm::cuda_stream_view stream)
{
  auto d_input      = cudf::column_device_view::create(input->view(), stream);
  auto merge_pairs  = initialize_merge_pairs_map(*d_input, stream);
  auto mp_table_map = initialize_mp_table_map(*d_input, stream);
  return std::make_unique<nvtext::bpe_merge_pairs::bpe_merge_pairs_impl>(
    std::move(input), std::move(d_input), std::move(merge_pairs), std::move(mp_table_map));
}

std::unique_ptr<bpe_merge_pairs::bpe_merge_pairs_impl> create_bpe_merge_pairs_impl(
  cudf::strings_column_view const& input,
  rmm::cuda_stream_view stream,
  rmm::mr::device_memory_resource* mr)
{
  auto pairs   = cudf::strings::split_record(input, cudf::string_scalar(" "), 1, stream, mr);
  auto content = pairs->release();
  return create_bpe_merge_pairs_impl(std::move(content.children.back()), stream);
}

}  // namespace

std::unique_ptr<bpe_merge_pairs> load_merge_pairs(cudf::strings_column_view const& merge_pairs,
                                                  rmm::cuda_stream_view stream,
                                                  rmm::mr::device_memory_resource* mr)
{
  CUDF_EXPECTS(!merge_pairs.is_empty(), "Merge pairs must not be empty");
  CUDF_EXPECTS(!merge_pairs.has_nulls(), "Merge pairs may not contain nulls");
  return std::make_unique<bpe_merge_pairs>(merge_pairs, stream, mr);
}

}  // namespace detail

std::unique_ptr<bpe_merge_pairs> load_merge_pairs(cudf::strings_column_view const& merge_pairs,
                                                  rmm::cuda_stream_view stream,
                                                  rmm::mr::device_memory_resource* mr)
{
  CUDF_FUNC_RANGE();
  return detail::load_merge_pairs(merge_pairs, stream, mr);
}

bpe_merge_pairs::bpe_merge_pairs_impl::bpe_merge_pairs_impl(
  std::unique_ptr<cudf::column>&& merge_pairs,
  std::unique_ptr<cudf::column_device_view, std::function<void(cudf::column_device_view*)>>&&
    d_merge_pairs,
  std::unique_ptr<detail::merge_pairs_map_type>&& merge_pairs_map,
  std::unique_ptr<detail::mp_table_map_type>&& mp_table_map)
  : merge_pairs(std::move(merge_pairs)),
    d_merge_pairs(std::move(d_merge_pairs)),
    merge_pairs_map(std::move(merge_pairs_map)),
    mp_table_map(std::move(mp_table_map))
{
}

bpe_merge_pairs::bpe_merge_pairs(std::unique_ptr<cudf::column>&& input,
                                 rmm::cuda_stream_view stream,
                                 rmm::mr::device_memory_resource*)
  : impl(detail::create_bpe_merge_pairs_impl(std::move(input), stream).release())
{
}

bpe_merge_pairs::bpe_merge_pairs(cudf::strings_column_view const& input,
                                 rmm::cuda_stream_view stream,
                                 rmm::mr::device_memory_resource* mr)
  : impl(detail::create_bpe_merge_pairs_impl(input, stream, mr).release())
{
}

bpe_merge_pairs::bpe_merge_pairs() = default;
bpe_merge_pairs::~bpe_merge_pairs() { delete impl; }

}  // namespace nvtext
