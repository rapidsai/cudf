#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2020, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <nvtext/edit_distance.hpp>

#include <cudf/column/column.hpp>
#include <cudf/column/column_device_view.cuh>
#include <cudf/column/column_factories.hpp>
#include <cudf/column/column_view.hpp>
#include <cudf/detail/nvtx/ranges.hpp>
#include <cudf/strings/string_view.cuh>
#include <cudf/strings/strings_column_view.hpp>

#include <rmm/cuda_stream_view.hpp>
#include <rmm/device_uvector.hpp>
#include <rmm/exec_policy.hpp>

#include <thrust/transform.h>
#include <thrust/transform_scan.h>

namespace nvtext {
namespace detail {
namespace {

/**
 * @brief Compute the edit-distance between two strings
 *
 * The temporary buffer must be able to hold 3 int16 values for each character
 * in the smaller of the two provided strings.
 *
 * @param d_str First string
 * @param d_tgt Second string
 * @param buffer Temporary memory buffer used for the calculation.
 * @return Edit distance value
 */
__device__ int32_t compute_distance(cudf::string_view const& d_str,
                                    cudf::string_view const& d_tgt,
                                    int16_t* buffer)
{
  auto const str_length = d_str.length();
  auto const tgt_length = d_tgt.length();
  if (str_length == 0) return tgt_length;
  if (tgt_length == 0) return str_length;

  auto itr_A = str_length < tgt_length ? d_str.begin() : d_tgt.begin();
  auto itr_B = str_length < tgt_length ? d_tgt.begin() : d_str.begin();
  // .first is min and .second is max
  auto const lengths = std::minmax(str_length, tgt_length);
  // setup compute buffer pointers
  auto line2 = buffer;
  auto line1 = line2 + lengths.first;
  auto line0 = line1 + lengths.first;
  // range is both lengths
  auto const range = lengths.first + lengths.second - 1;
  for (cudf::size_type i = 0; i < range; ++i) {
    auto tmp = line2;
    line2    = line1;
    line1    = line0;
    line0    = tmp;
    // checking pairs of characters
    for (int x = (i < lengths.second ? 0 : i - lengths.second + 1);
         (x < lengths.first) && (x < i + 1);
         ++x) {
      int const y = i - x;
      itr_A += (x - itr_A.position());  // point to next
      itr_B += (y - itr_B.position());  // characters to check
      int16_t const w =
        (((x > 0) && (y > 0)) ? line2[x - 1] : static_cast<int16_t>(std::max(x, y))) +
        static_cast<int16_t>(*itr_A != *itr_B);  // add 1 if characters do not match
      int16_t const u = (y > 0 ? line1[x] : x + 1) + 1;
      int16_t const v = (x > 0 ? line1[x - 1] : y + 1) + 1;
      // store min(u,v,w)
      line0[x] = std::min(std::min(u, v), w);
    }
  }
  return static_cast<int32_t>(line0[lengths.first - 1]);
}

/**
 * @brief Compute the Levenshtein distance for each string.
 *
 * Documentation here: https://www.cuelogic.com/blog/the-levenshtein-algorithm
 * And here: https://en.wikipedia.org/wiki/Levenshtein_distances
 */
struct edit_distance_levenshtein_algorithm {
  cudf::column_device_view d_strings;  // computing these
  cudf::column_device_view d_targets;  // against these;
  int16_t* d_buffer;                   // compute buffer for each string
  int32_t* d_results;                  // input is buffer offset; output is edit distance

  __device__ void operator()(cudf::size_type idx)
  {
    auto d_str =
      d_strings.is_null(idx) ? cudf::string_view{} : d_strings.element<cudf::string_view>(idx);
    auto d_tgt = [&] __device__ {  // d_targets is also allowed to have only one entry
      if (d_targets.is_null(idx)) return cudf::string_view{};
      return d_targets.size() == 1 ? d_targets.element<cudf::string_view>(0)
                                   : d_targets.element<cudf::string_view>(idx);
    }();
    d_results[idx] = compute_distance(d_str, d_tgt, d_buffer + d_results[idx]);
  }
};

struct edit_distance_matrix_levenshtein_algorithm {
  cudf::column_device_view d_strings;  // computing these against itself
  int16_t* d_buffer;                   // compute buffer for each string
  int32_t const* d_offsets;            // locate sub-buffer for each string
  int32_t* d_results;                  // edit distance values

  __device__ void operator()(cudf::size_type idx)
  {
    auto const strings_count = d_strings.size();
    auto const row           = idx / strings_count;
    auto const col           = idx % strings_count;
    if (row > col) return;  // bottom half is computed with the top half of matrix
    cudf::string_view d_str1 =
      d_strings.is_null(row) ? cudf::string_view{} : d_strings.element<cudf::string_view>(row);
    cudf::string_view d_str2 =
      d_strings.is_null(col) ? cudf::string_view{} : d_strings.element<cudf::string_view>(col);
    auto work_buffer       = d_buffer + d_offsets[idx - ((row + 1) * (row + 2)) / 2];
    int32_t const distance = (row == col) ? 0 : compute_distance(d_str1, d_str2, work_buffer);
    d_results[idx]         = distance;                // top half of matrix
    d_results[col * strings_count + row] = distance;  // bottom half of matrix
  }
};

}  // namespace

/**
 * @copydoc nvtext::edit_distance
 */
std::unique_ptr<cudf::column> edit_distance(cudf::strings_column_view const& strings,
                                            cudf::strings_column_view const& targets,
                                            rmm::cuda_stream_view stream,
                                            rmm::mr::device_memory_resource* mr)
{
  cudf::size_type strings_count = strings.size();
  if (strings_count == 0) return cudf::make_empty_column(cudf::data_type{cudf::type_id::INT32});
  if (targets.size() > 1)
    CUDF_EXPECTS(strings_count == targets.size(), "targets.size() must equal strings.size()");

  // create device columns from the input columns
  auto strings_column = cudf::column_device_view::create(strings.parent(), stream);
  auto d_strings      = *strings_column;
  auto targets_column = cudf::column_device_view::create(targets.parent(), stream);
  auto d_targets      = *targets_column;

  // calculate the size of the compute-buffer;
  // we can use the output column buffer to hold the size/offset values temporarily
  auto results   = cudf::make_fixed_width_column(cudf::data_type{cudf::type_id::INT32},
                                               strings_count,
                                               rmm::device_buffer{0, stream, mr},
                                               0,
                                               stream,
                                               mr);
  auto d_results = results->mutable_view().data<int32_t>();

  thrust::transform(rmm::exec_policy(stream),
                    thrust::make_counting_iterator<cudf::size_type>(0),
                    thrust::make_counting_iterator<cudf::size_type>(strings_count),
                    d_results,
                    [d_strings, d_targets] __device__(auto idx) {
                      if (d_strings.is_null(idx) || d_targets.is_null(idx)) return int32_t{0};
                      auto d_str = d_strings.element<cudf::string_view>(idx);
                      auto d_tgt = d_targets.size() == 1
                                     ? d_targets.element<cudf::string_view>(0)
                                     : d_targets.element<cudf::string_view>(idx);
                      // just need 3 int16's for each character of the shorter string
                      return static_cast<int32_t>(std::min(d_str.length(), d_tgt.length()) * 3);
                    });

  // get the total size of the temporary compute buffer
  size_t compute_size =
    thrust::reduce(rmm::exec_policy(stream), d_results, d_results + strings_count, size_t{0});
  // convert sizes to offsets in-place
  thrust::exclusive_scan(rmm::exec_policy(stream), d_results, d_results + strings_count, d_results);
  // create the temporary compute buffer
  rmm::device_uvector<int16_t> compute_buffer(compute_size, stream);
  auto d_buffer = compute_buffer.data();

  // compute the edit distance into the output column in-place
  // - on input, d_results is the offset to the working section of d_buffer for each row
  // - on output, d_results is the calculated edit distance for that row
  thrust::for_each_n(
    rmm::exec_policy(stream),
    thrust::make_counting_iterator<cudf::size_type>(0),
    strings_count,
    edit_distance_levenshtein_algorithm{d_strings, d_targets, d_buffer, d_results});
  return results;
}

/**
 * @copydoc nvtext::edit_distance_matrix
 */
std::unique_ptr<cudf::column> edit_distance_matrix(cudf::strings_column_view const& strings,
                                                   rmm::cuda_stream_view stream,
                                                   rmm::mr::device_memory_resource* mr)
{
  cudf::size_type strings_count = strings.size();
  if (strings_count == 0) return cudf::make_empty_column(cudf::data_type{cudf::type_id::INT32});
  CUDF_EXPECTS(strings_count > 1, "the input strings must include at least 2 strings");
  CUDF_EXPECTS(static_cast<size_t>(strings_count) * static_cast<size_t>(strings_count) <
                 std::numeric_limits<int32_t>().max(),
               "too many strings to create the output column");

  // create device column of the input strings column
  auto strings_column = cudf::column_device_view::create(strings.parent(), stream);
  auto d_strings      = *strings_column;

  // Calculate the size of the compute-buffer.
  // We only need memory for half the size of the output matrix since the edit distance calculation
  // is commutative -- `distance(strings[i],strings[j]) == distance(strings[j],strings[i])`
  cudf::size_type n_upper = (strings_count * (strings_count - 1)) / 2;
  rmm::device_uvector<cudf::size_type> offsets(n_upper, stream);
  auto d_offsets = offsets.data();
  CUDA_TRY(hipMemsetAsync(d_offsets, 0, n_upper * sizeof(cudf::size_type), stream.value()));
  thrust::for_each_n(
    rmm::exec_policy(stream),
    thrust::make_counting_iterator<cudf::size_type>(0),
    strings_count * strings_count,
    [d_strings, d_offsets, strings_count] __device__(cudf::size_type idx) {
      auto const row = idx / strings_count;
      auto const col = idx % strings_count;
      if (row >= col) return;  // compute only the top half
      cudf::string_view const d_str1 =
        d_strings.is_null(row) ? cudf::string_view{} : d_strings.element<cudf::string_view>(row);
      cudf::string_view const d_str2 =
        d_strings.is_null(col) ? cudf::string_view{} : d_strings.element<cudf::string_view>(col);
      if (d_str1.empty() || d_str2.empty()) return;
      // the temp size needed is 3 int16s per character of the shorter string
      d_offsets[idx - ((row + 1) * (row + 2)) / 2] = std::min(d_str1.length(), d_str2.length()) * 3;
    });

  // get the total size for the compute buffer
  size_t compute_size =
    thrust::reduce(rmm::exec_policy(stream), offsets.begin(), offsets.end(), size_t{0});
  // convert sizes to offsets in-place
  thrust::exclusive_scan(rmm::exec_policy(stream), offsets.begin(), offsets.end(), offsets.begin());
  // create the compute buffer
  rmm::device_uvector<int16_t> compute_buffer(compute_size, stream);
  auto d_buffer = compute_buffer.data();

  // compute the edit distance into the output column
  auto results   = cudf::make_fixed_width_column(cudf::data_type{cudf::type_id::INT32},
                                               strings_count * strings_count,
                                               rmm::device_buffer{0, stream, mr},
                                               0,
                                               stream,
                                               mr);
  auto d_results = results->mutable_view().data<int32_t>();
  thrust::for_each_n(
    rmm::exec_policy(stream),
    thrust::make_counting_iterator<cudf::size_type>(0),
    strings_count * strings_count,
    edit_distance_matrix_levenshtein_algorithm{d_strings, d_buffer, d_offsets, d_results});

  // build a lists column of the results
  auto offsets_column = cudf::make_fixed_width_column(cudf::data_type{cudf::type_id::INT32},
                                                      strings_count + 1,
                                                      rmm::device_buffer{0, stream, mr},
                                                      0,
                                                      stream,
                                                      mr);
  thrust::transform_exclusive_scan(
    rmm::exec_policy(stream),
    thrust::make_counting_iterator<int32_t>(0),
    thrust::make_counting_iterator<int32_t>(strings_count + 1),
    offsets_column->mutable_view().data<int32_t>(),
    [strings_count] __device__(auto idx) { return strings_count; },
    int32_t{0},
    thrust::plus<int32_t>());
  return cudf::make_lists_column(strings_count,
                                 std::move(offsets_column),
                                 std::move(results),
                                 0,  // no nulls
                                 rmm::device_buffer{0, stream, mr},
                                 stream,
                                 mr);
}

}  // namespace detail

// external APIs

/**
 * @copydoc nvtext::edit_distance
 */
std::unique_ptr<cudf::column> edit_distance(cudf::strings_column_view const& strings,
                                            cudf::strings_column_view const& targets,
                                            rmm::mr::device_memory_resource* mr)
{
  CUDF_FUNC_RANGE();
  return detail::edit_distance(strings, targets, rmm::cuda_stream_default, mr);
}

/**
 * @copydoc nvtext::edit_distance_matrix
 */
std::unique_ptr<cudf::column> edit_distance_matrix(cudf::strings_column_view const& strings,
                                                   rmm::mr::device_memory_resource* mr)
{
  CUDF_FUNC_RANGE();
  return detail::edit_distance_matrix(strings, rmm::cuda_stream_default, mr);
}

}  // namespace nvtext
