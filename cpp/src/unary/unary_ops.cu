#include "hip/hip_runtime.h"
/*
 *
 * Code edits and additions
 * 		Copyright 2018 Rommel Quintanilla <rommel@blazingdb.com>
 */

#include <cmath>
#include <algorithm>

#include <thrust/copy.h>
#include <thrust/execution_policy.h>

#include "cudf.h"
#include "utilities/cudf_utils.h"
#include "utilities/error_utils.h"
#include "rmm/thrust_rmm_allocator.h"

template<typename T, typename Tout, typename F>
__global__
void gpu_unary_op(const T *data, const gdf_valid_type *valid,
                  gdf_size_type size, Tout *results, F functor) {
    int tid = threadIdx.x;
    int blkid = blockIdx.x;
    int blksz = blockDim.x;
    int gridsz = gridDim.x;

    int start = tid + blkid * blksz;
    int step = blksz * gridsz;
    if ( valid ) {  // has valid mask
        for (int i=start; i<size; i+=step) {
            if ( gdf_is_valid(valid, i) )
                results[i] = functor.apply(data[i]);
        }
    } else {        // no valid mask
        for (int i=start; i<size; i+=step) {
            results[i] = functor.apply(data[i]);
        }
    }
}

template<typename T, typename Tout, typename F>
struct UnaryOp {
    static
    gdf_error launch(gdf_column *input, gdf_column *output) {

        // Return immediately for empty inputs
        if((0==input->size))
        {
          return GDF_SUCCESS;
        }

        /* check for size of the columns */
        if (input->size != output->size) {
            return GDF_COLUMN_SIZE_MISMATCH;
        }

        // find optimal blocksize
        int mingridsize, blocksize;
        CUDA_TRY(
            hipOccupancyMaxPotentialBlockSize(&mingridsize, &blocksize,
                                               gpu_unary_op<T, Tout, F>)
        );
        // find needed gridsize
        int neededgridsize = (input->size + blocksize - 1) / blocksize;
        int gridsize = std::min(neededgridsize, mingridsize);

        F functor;
        gpu_unary_op<<<gridsize, blocksize>>>(
            // input
            (const T*)input->data, input->valid, input->size,
            // output
            (Tout*)output->data,
            // action
            functor
        );

        CUDA_CHECK_LAST();
        return GDF_SUCCESS;
    }
};


template<typename T, typename F>
struct MathOp {
    static
    gdf_error launch(gdf_column *input, gdf_column *output) {
        return UnaryOp<T, T, F>::launch(input, output);
    }
};


#define DEF_UNARY_OP_REAL(F)                                        \
gdf_error F##_generic(gdf_column *input, gdf_column *output) {      \
    switch ( input->dtype ) {                                       \
    case GDF_FLOAT32: return F##_f32(input, output);                \
    case GDF_FLOAT64: return F##_f64(input, output);                \
    default: return GDF_UNSUPPORTED_DTYPE;                          \
    }                                                               \
}

#define DEF_CAST_OP(TO)                                                       \
gdf_error gdf_cast_generic_to_##TO(gdf_column *input, gdf_column *output) {   \
    switch ( input->dtype ) {                                                 \
    case      GDF_INT8: return gdf_cast_i8_to_##TO(input, output);            \
    case     GDF_INT32: return gdf_cast_i32_to_##TO(input, output);           \
    case     GDF_INT64: return gdf_cast_i64_to_##TO(input, output);           \
    case   GDF_FLOAT32: return gdf_cast_f32_to_##TO(input, output);           \
    case   GDF_FLOAT64: return gdf_cast_f64_to_##TO(input, output);           \
    case    GDF_DATE32: return gdf_cast_date32_to_##TO(input, output);        \
    case    GDF_DATE64: return gdf_cast_date64_to_##TO(input, output);        \
    case GDF_TIMESTAMP: return gdf_cast_timestamp_to_##TO(input, output);     \
    default: return GDF_UNSUPPORTED_DTYPE;                                    \
    }                                                                         \
}

#define DEF_CAST_OP_TS(TO)                                                                          \
gdf_error gdf_cast_generic_to_##TO(gdf_column *input, gdf_column *output, gdf_time_unit time_unit) {\
    switch ( input->dtype ) {                                                                       \
    case      GDF_INT8: return gdf_cast_i8_to_##TO(input, output, time_unit);                       \
    case     GDF_INT32: return gdf_cast_i32_to_##TO(input, output, time_unit);                      \
    case     GDF_INT64: return gdf_cast_i64_to_##TO(input, output, time_unit);                      \
    case   GDF_FLOAT32: return gdf_cast_f32_to_##TO(input, output, time_unit);                      \
    case   GDF_FLOAT64: return gdf_cast_f64_to_##TO(input, output, time_unit);                      \
    case    GDF_DATE32: return gdf_cast_date32_to_##TO(input, output, time_unit);                   \
    case    GDF_DATE64: return gdf_cast_date64_to_##TO(input, output, time_unit);                   \
    case GDF_TIMESTAMP: return gdf_cast_timestamp_to_##TO(input, output, time_unit);                \
    default: return GDF_UNSUPPORTED_DTYPE;                                                          \
    }                                                                                               \
}

// trig functions

template<typename T>
struct DeviceSin {
    __device__
    T apply(T data) {
        return std::sin(data);
    }
};

template<typename T>
struct DeviceCos {
    __device__
    T apply(T data) {
        return std::cos(data);
    }
};

template<typename T>
struct DeviceTan {
    __device__
    T apply(T data) {
        return std::tan(data);
    }
};

template<typename T>
struct DeviceArcSin {
    __device__
    T apply(T data) {
        return std::asin(data);
    }
};

template<typename T>
struct DeviceArcCos {
    __device__
    T apply(T data) {
        return std::acos(data);
    }
};

template<typename T>
struct DeviceArcTan {
    __device__
    T apply(T data) {
        return std::atan(data);
    }
};

DEF_UNARY_OP_REAL(gdf_sin)

gdf_error gdf_sin_f32(gdf_column *input, gdf_column *output) {
    return MathOp<float, DeviceSin<float> >::launch(input, output);
}

gdf_error gdf_sin_f64(gdf_column *input, gdf_column *output) {
    return MathOp<double, DeviceSin<double> >::launch(input, output);
}

DEF_UNARY_OP_REAL(gdf_cos)

gdf_error gdf_cos_f32(gdf_column *input, gdf_column *output) {
    return MathOp<float, DeviceCos<float> >::launch(input, output);
}

gdf_error gdf_cos_f64(gdf_column *input, gdf_column *output) {
    return MathOp<double, DeviceCos<double> >::launch(input, output);
}

DEF_UNARY_OP_REAL(gdf_tan)

gdf_error gdf_tan_f32(gdf_column *input, gdf_column *output) {
    return MathOp<float, DeviceTan<float> >::launch(input, output);
}

gdf_error gdf_tan_f64(gdf_column *input, gdf_column *output) {
    return MathOp<double, DeviceTan<double> >::launch(input, output);
}

DEF_UNARY_OP_REAL(gdf_asin)

gdf_error gdf_asin_f32(gdf_column *input, gdf_column *output) {
    return MathOp<float, DeviceArcSin<float> >::launch(input, output);
}

gdf_error gdf_asin_f64(gdf_column *input, gdf_column *output) {
    return MathOp<double, DeviceArcSin<double> >::launch(input, output);
}

DEF_UNARY_OP_REAL(gdf_acos)

gdf_error gdf_acos_f32(gdf_column *input, gdf_column *output) {
    return MathOp<float, DeviceArcCos<float> >::launch(input, output);
}

gdf_error gdf_acos_f64(gdf_column *input, gdf_column *output) {
    return MathOp<double, DeviceArcCos<double> >::launch(input, output);
}

DEF_UNARY_OP_REAL(gdf_atan)

gdf_error gdf_atan_f32(gdf_column *input, gdf_column *output) {
    return MathOp<float, DeviceArcTan<float> >::launch(input, output);
}

gdf_error gdf_atan_f64(gdf_column *input, gdf_column *output) {
    return MathOp<double, DeviceArcTan<double> >::launch(input, output);
}

// exponential functions

template<typename T>
struct DeviceExp {
    __device__
    T apply(T data) {
        return std::exp(data);
    }
};

template<typename T>
struct DeviceLog {
    __device__
    T apply(T data) {
        return std::log(data);
    }
};

DEF_UNARY_OP_REAL(gdf_exp)

gdf_error gdf_exp_f32(gdf_column *input, gdf_column *output) {
    return MathOp<float, DeviceExp<float> >::launch(input, output);
}

gdf_error gdf_exp_f64(gdf_column *input, gdf_column *output) {
    return MathOp<double, DeviceExp<double> >::launch(input, output);
}

DEF_UNARY_OP_REAL(gdf_log)

gdf_error gdf_log_f32(gdf_column *input, gdf_column *output) {
    return MathOp<float, DeviceLog<float> >::launch(input, output);
}

gdf_error gdf_log_f64(gdf_column *input, gdf_column *output) {
    return MathOp<double, DeviceLog<double> >::launch(input, output);
}

// exponential functions

template<typename T>
struct DeviceSqrt {
    __device__
    T apply(T data) {
        return std::sqrt(data);
    }
};

DEF_UNARY_OP_REAL(gdf_sqrt)

gdf_error gdf_sqrt_f32(gdf_column *input, gdf_column *output) {
    return MathOp<float, DeviceSqrt<float> >::launch(input, output);
}

gdf_error gdf_sqrt_f64(gdf_column *input, gdf_column *output) {
    return MathOp<double, DeviceSqrt<double> >::launch(input, output);
}

// rounding functions

template<typename T>
struct DeviceCeil {
    __device__
    T apply(T data) {
        return std::ceil(data);
    }
};

template<typename T>
struct DeviceFloor {
    __device__
    T apply(T data) {
        return std::floor(data);
    }
};

DEF_UNARY_OP_REAL(gdf_ceil)

gdf_error gdf_ceil_f32(gdf_column *input, gdf_column *output) {
    return MathOp<float, DeviceCeil<float> >::launch(input, output);
}

gdf_error gdf_ceil_f64(gdf_column *input, gdf_column *output) {
    return MathOp<double, DeviceCeil<double> >::launch(input, output);
}

DEF_UNARY_OP_REAL(gdf_floor)

gdf_error gdf_floor_f32(gdf_column *input, gdf_column *output) {
    return MathOp<float, DeviceFloor<float> >::launch(input, output);
}

gdf_error gdf_floor_f64(gdf_column *input, gdf_column *output) {
    return MathOp<double, DeviceFloor<double> >::launch(input, output);
}


// casting

template<typename From, typename To>
struct DeviceCast {
    __device__
    To apply(From data) {
        return (To)data;
    }
};

template<typename From, typename To, int64_t units_factor>
struct UpCasting {
    __device__
    To apply(From data) {
        return (To)(data*units_factor);
    }
};

template<typename From, typename To, int64_t units_factor>
struct DownCasting {
    __device__
    To apply(From data) {
        return (To)((data-(units_factor-1)*(data<0))/units_factor); //ceiling only when data is negative
    }
};

// Castings are differentiate between physical and logical ones.
// In physical casting only change the physical representation, for example from GDF_FLOAT32 (float) to GDF_FLOAT64 (double)
// on the other hand, casting between date timestamps needs also perform some calculations according to the time unit:
// - when the source or destination datatype is GDF_DATE32, the value is multiplied or divided by the amount of timeunits by day
// - when datatypes are timestamps, the value is multiplied or divided according to the S.I. nano 10^-9, micro 10^-6, milli 10^-3
// No calculation is necessary when casting between GDF_DATE64 and GDF_TIMESTAMP (with ms as time unit), because are logically and physically the same thing

#define DEF_CAST_IMPL(VFROM, VTO, TFROM, TTO, LTFROM, LTO)                                                      \
gdf_error gdf_cast_##VFROM##_to_##VTO(gdf_column *input, gdf_column *output) {                                  \
    GDF_REQUIRE(input->dtype == LTFROM, GDF_UNSUPPORTED_DTYPE);                                                 \
                                                                                                                \
                                                                                                                \
    output->dtype = LTO;                                                                                        \
    if (input->valid && output->valid) {                                                                        \
        gdf_size_type num_chars_bitmask = gdf_get_num_chars_bitmask(input->size);                               \
        thrust::copy(rmm::exec_policy(hipStream_t{0}), input->valid, input->valid + num_chars_bitmask, output->valid);            \
    }                                                                                                           \
                                                                                                                \
    /* Handling datetime logical castings */                                                                    \
    if( LTFROM == GDF_DATE64 && LTO == GDF_DATE32 )                                                             \
        return UnaryOp<TFROM, TTO, DownCasting<TFROM, TTO, 86400000> >::launch(input, output);                  \
    else if( LTFROM == GDF_DATE32 && LTO == GDF_DATE64 )                                                        \
        return UnaryOp<TFROM, TTO, UpCasting<TFROM, TTO, 86400000> >::launch(input, output);                    \
    else if( ( LTFROM == GDF_TIMESTAMP && input->dtype_info.time_unit == TIME_UNIT_s ) && LTO == GDF_DATE32 )   \
        return UnaryOp<TFROM, TTO, DownCasting<TFROM, TTO, 86400> >::launch(input, output);                     \
    else if( ( LTFROM == GDF_TIMESTAMP && input->dtype_info.time_unit == TIME_UNIT_ms ) && LTO == GDF_DATE32 )  \
        return UnaryOp<TFROM, TTO, DownCasting<TFROM, TTO, 86400000> >::launch(input, output);                  \
    else if( ( LTFROM == GDF_TIMESTAMP && input->dtype_info.time_unit == TIME_UNIT_us ) && LTO == GDF_DATE32 )  \
        return UnaryOp<TFROM, TTO, DownCasting<TFROM, TTO, 86400000000> >::launch(input, output);               \
    else if( ( LTFROM == GDF_TIMESTAMP && input->dtype_info.time_unit == TIME_UNIT_ns ) && LTO == GDF_DATE32 )  \
        return UnaryOp<TFROM, TTO, DownCasting<TFROM, TTO, 86400000000000> >::launch(input, output);            \
    else if( ( LTFROM == GDF_TIMESTAMP && input->dtype_info.time_unit == TIME_UNIT_s ) && LTO == GDF_DATE64 )   \
        return UnaryOp<TFROM, TTO, UpCasting<TFROM, TTO, 1000> >::launch(input, output);                        \
    else if( ( LTFROM == GDF_TIMESTAMP && input->dtype_info.time_unit == TIME_UNIT_us ) && LTO == GDF_DATE64 )  \
        return UnaryOp<TFROM, TTO, DownCasting<TFROM, TTO, 1000> >::launch(input, output);                      \
    else if( ( LTFROM == GDF_TIMESTAMP && input->dtype_info.time_unit == TIME_UNIT_ns ) && LTO == GDF_DATE64 )  \
        return UnaryOp<TFROM, TTO, DownCasting<TFROM, TTO, 1000000> >::launch(input, output);                   \
    /* Handling only physical castings */                                                                       \
    return UnaryOp<TFROM, TTO, DeviceCast<TFROM, TTO> >::launch(input, output);                                 \
}

// Castings functions where Timestamp is the destination type
#define DEF_CAST_IMPL_TS(VFROM, VTO, TFROM, TTO, LTFROM, LTO)                                           \
gdf_error gdf_cast_##VFROM##_to_##VTO(gdf_column *input, gdf_column *output, gdf_time_unit time_unit) { \
    GDF_REQUIRE(input->dtype == LTFROM, GDF_UNSUPPORTED_DTYPE);                                         \
                                                                                                        \
                                                                                                        \
    output->dtype = LTO;                                                                                \
    output->dtype_info.time_unit = time_unit;                                                           \
    if (input->valid && output->valid) {                                                                \
        gdf_size_type num_chars_bitmask = gdf_get_num_chars_bitmask(input->size);                       \
        thrust::copy(rmm::exec_policy(hipStream_t{0}), input->valid, input->valid + num_chars_bitmask, output->valid);    \
    }                                                                                                   \
                                                                                                        \
    /* Handling datetime logical castings */                                                            \
    if( LTFROM == GDF_DATE32 && ( LTO == GDF_TIMESTAMP && time_unit == TIME_UNIT_s ) )                  \
        return UnaryOp<TFROM, TTO, UpCasting<TFROM, TTO, 86400> >::launch(input, output);               \
    else if( LTFROM == GDF_DATE32 && ( LTO == GDF_TIMESTAMP && time_unit == TIME_UNIT_ms ) )            \
        return UnaryOp<TFROM, TTO, UpCasting<TFROM, TTO, 86400000> >::launch(input, output);            \
    else if( LTFROM == GDF_DATE32 && ( LTO == GDF_TIMESTAMP && time_unit == TIME_UNIT_us ) )            \
        return UnaryOp<TFROM, TTO, UpCasting<TFROM, TTO, 86400000000> >::launch(input, output);         \
    else if( LTFROM == GDF_DATE32 && ( LTO == GDF_TIMESTAMP && time_unit == TIME_UNIT_ns ) )            \
        return UnaryOp<TFROM, TTO, UpCasting<TFROM, TTO, 86400000000000> >::launch(input, output);      \
    else if( LTFROM == GDF_DATE64 && LTO == GDF_TIMESTAMP && time_unit == TIME_UNIT_us)                 \
        return UnaryOp<TFROM, TTO, UpCasting<TFROM, TTO, 1000> >::launch(input, output);                \
    else if( LTFROM == GDF_DATE64 && LTO == GDF_TIMESTAMP && time_unit == TIME_UNIT_s)                  \
        return UnaryOp<TFROM, TTO, DownCasting<TFROM, TTO, 1000> >::launch(input, output);              \
    else if( LTFROM == GDF_DATE64 && LTO == GDF_TIMESTAMP && time_unit == TIME_UNIT_ns)                 \
        return UnaryOp<TFROM, TTO, UpCasting<TFROM, TTO, 1000000> >::launch(input, output);             \
    else if( LTFROM == GDF_TIMESTAMP && LTO == GDF_TIMESTAMP )                                          \
    {                                                                                                   \
        if( input->dtype_info.time_unit == TIME_UNIT_s && time_unit == TIME_UNIT_ms )                   \
            return UnaryOp<TFROM, TTO, UpCasting<TFROM, TTO, 1000> >::launch(input, output);            \
        else if( input->dtype_info.time_unit == TIME_UNIT_ms && time_unit == TIME_UNIT_s )              \
            return UnaryOp<TFROM, TTO, DownCasting<TFROM, TTO, 1000> >::launch(input, output);          \
        else if( input->dtype_info.time_unit == TIME_UNIT_s && time_unit == TIME_UNIT_us )              \
            return UnaryOp<TFROM, TTO, UpCasting<TFROM, TTO, 1000000> >::launch(input, output);         \
        else if( input->dtype_info.time_unit == TIME_UNIT_us && time_unit == TIME_UNIT_s )              \
            return UnaryOp<TFROM, TTO, DownCasting<TFROM, TTO, 1000000> >::launch(input, output);       \
        else if( input->dtype_info.time_unit == TIME_UNIT_s && time_unit == TIME_UNIT_ns )              \
            return UnaryOp<TFROM, TTO, UpCasting<TFROM, TTO, 1000000000> >::launch(input, output);      \
        else if( input->dtype_info.time_unit == TIME_UNIT_ns && time_unit == TIME_UNIT_s )              \
            return UnaryOp<TFROM, TTO, DownCasting<TFROM, TTO, 1000000000> >::launch(input, output);    \
        else if( input->dtype_info.time_unit == TIME_UNIT_us && time_unit == TIME_UNIT_ns )             \
            return UnaryOp<TFROM, TTO, UpCasting<TFROM, TTO, 1000> >::launch(input, output);            \
        else if( input->dtype_info.time_unit == TIME_UNIT_ns && time_unit == TIME_UNIT_us )             \
            return UnaryOp<TFROM, TTO, DownCasting<TFROM, TTO, 1000> >::launch(input, output);          \
        else if( input->dtype_info.time_unit == TIME_UNIT_ms && time_unit == TIME_UNIT_ns )             \
            return UnaryOp<TFROM, TTO, UpCasting<TFROM, TTO, 1000000> >::launch(input, output);         \
        else if( input->dtype_info.time_unit == TIME_UNIT_ns && time_unit == TIME_UNIT_ms )             \
            return UnaryOp<TFROM, TTO, DownCasting<TFROM, TTO, 1000000> >::launch(input, output);       \
        else if( input->dtype_info.time_unit == TIME_UNIT_us && time_unit == TIME_UNIT_ms )             \
            return UnaryOp<TFROM, TTO, DownCasting<TFROM, TTO, 1000> >::launch(input, output);          \
        else if( input->dtype_info.time_unit == TIME_UNIT_ms && time_unit == TIME_UNIT_us )             \
            return UnaryOp<TFROM, TTO, UpCasting<TFROM, TTO, 1000> >::launch(input, output);            \
    }                                                                                                   \
    /* Handling only physical castings */                                                               \
    return UnaryOp<TFROM, TTO, DeviceCast<TFROM, TTO> >::launch(input, output);                         \
}

#define DEF_CAST_IMPL_TEMPLATE(ABREV, PHYSICAL_TYPE, LOGICAL_TYPE)                    \
DEF_CAST_OP(ABREV)                                                                    \
DEF_CAST_IMPL(i8,        ABREV,  int8_t, PHYSICAL_TYPE, GDF_INT8,       LOGICAL_TYPE) \
DEF_CAST_IMPL(i32,       ABREV, int32_t, PHYSICAL_TYPE, GDF_INT32,      LOGICAL_TYPE) \
DEF_CAST_IMPL(i64,       ABREV, int64_t, PHYSICAL_TYPE, GDF_INT64,      LOGICAL_TYPE) \
DEF_CAST_IMPL(f32,       ABREV,   float, PHYSICAL_TYPE, GDF_FLOAT32,    LOGICAL_TYPE) \
DEF_CAST_IMPL(f64,       ABREV,  double, PHYSICAL_TYPE, GDF_FLOAT64,    LOGICAL_TYPE) \
DEF_CAST_IMPL(date32,    ABREV, int32_t, PHYSICAL_TYPE, GDF_DATE32,     LOGICAL_TYPE) \
DEF_CAST_IMPL(date64,    ABREV, int64_t, PHYSICAL_TYPE, GDF_DATE64,     LOGICAL_TYPE) \
DEF_CAST_IMPL(timestamp, ABREV, int64_t, PHYSICAL_TYPE, GDF_TIMESTAMP,  LOGICAL_TYPE)

#define DEF_CAST_IMPL_TEMPLATE_TS(ABREV, PHYSICAL_TYPE, LOGICAL_TYPE)                    \
DEF_CAST_OP_TS(ABREV)                                                                    \
DEF_CAST_IMPL_TS(i8,        ABREV,  int8_t, PHYSICAL_TYPE, GDF_INT8,       LOGICAL_TYPE) \
DEF_CAST_IMPL_TS(i32,       ABREV, int32_t, PHYSICAL_TYPE, GDF_INT32,      LOGICAL_TYPE) \
DEF_CAST_IMPL_TS(i64,       ABREV, int64_t, PHYSICAL_TYPE, GDF_INT64,      LOGICAL_TYPE) \
DEF_CAST_IMPL_TS(f32,       ABREV,   float, PHYSICAL_TYPE, GDF_FLOAT32,    LOGICAL_TYPE) \
DEF_CAST_IMPL_TS(f64,       ABREV,  double, PHYSICAL_TYPE, GDF_FLOAT64,    LOGICAL_TYPE) \
DEF_CAST_IMPL_TS(date32,    ABREV, int32_t, PHYSICAL_TYPE, GDF_DATE32,     LOGICAL_TYPE) \
DEF_CAST_IMPL_TS(date64,    ABREV, int64_t, PHYSICAL_TYPE, GDF_DATE64,     LOGICAL_TYPE) \
DEF_CAST_IMPL_TS(timestamp, ABREV, int64_t, PHYSICAL_TYPE, GDF_TIMESTAMP,  LOGICAL_TYPE)

DEF_CAST_IMPL_TEMPLATE(f32, float, GDF_FLOAT32)
DEF_CAST_IMPL_TEMPLATE(f64, double, GDF_FLOAT64)
DEF_CAST_IMPL_TEMPLATE(i8, int8_t, GDF_INT8)
DEF_CAST_IMPL_TEMPLATE(i32, int32_t, GDF_INT32)
DEF_CAST_IMPL_TEMPLATE(i64, int64_t, GDF_INT64)
DEF_CAST_IMPL_TEMPLATE(date32, int32_t, GDF_DATE32)
DEF_CAST_IMPL_TEMPLATE(date64, int64_t, GDF_DATE64)
DEF_CAST_IMPL_TEMPLATE_TS(timestamp, int64_t, GDF_TIMESTAMP)
