/*
 * Copyright (c) 2019, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <cudf/cudf.h>
#include <cudf/types.hpp>
#include <cudf/utilities/legacy/type_dispatcher.hpp>
#include <utilities/cuda_utils.hpp>
#include <utilities/column_utils.hpp>
#include <bitmask/legacy/bit_mask.cuh>
#include <cudf/legacy/filling.hpp>

using bit_mask::bit_mask_t;

namespace cudf {

namespace detail {

gdf_column null_op(gdf_column const& input, bool nulls_are_false = true, hipStream_t stream = 0) {
    auto output = cudf::allocate_column(GDF_BOOL8, input.size, false, 
		  gdf_dtype_extra_info{}, stream);

    if (not cudf::is_nullable(input)) {
	gdf_scalar value {nulls_are_false, GDF_BOOL8, true}; 
	cudf::fill(&output, value, 0, output.size);
    } else {
        const bit_mask_t* __restrict__ typed_input_valid = reinterpret_cast<bit_mask_t*>(input.valid);

        thrust::transform(rmm::exec_policy(stream)->on(stream),
                          thrust::make_counting_iterator(static_cast<gdf_size_type>(0)),
                          thrust::make_counting_iterator(static_cast<gdf_size_type>(input.size)),
                          static_cast<bool*>(output.data),
                          [=]__device__(auto index){
                              return (nulls_are_false ==
                                      bit_mask::is_valid(typed_input_valid, index));
                          });
    }

    return output;
}
}// detail

gdf_column is_null(gdf_column const& input) {
    return detail::null_op(input, false, 0);
}

gdf_column is_not_null(gdf_column const& input) {
    return detail::null_op(input, true, 0);
}

}// cudf
