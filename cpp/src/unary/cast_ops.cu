#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <cudf/column/column.hpp>
#include <cudf/detail/nvtx/ranges.hpp>
#include <cudf/detail/unary.hpp>
#include <cudf/null_mask.hpp>
#include <cudf/unary.hpp>
#include <cudf/utilities/traits.hpp>

#include <rmm/thrust_rmm_allocator.h>

namespace cudf {
namespace detail {
template <typename _TargetT>
struct unary_cast {
  template <typename SourceT,
            typename TargetT                                          = _TargetT,
            typename std::enable_if_t<(cudf::is_numeric<SourceT>() &&
                                       cudf::is_numeric<TargetT>())>* = nullptr>
  CUDA_DEVICE_CALLABLE TargetT operator()(SourceT const element)
  {
    return static_cast<TargetT>(element);
  }

  template <typename SourceT,
            typename TargetT                                            = _TargetT,
            typename std::enable_if_t<(cudf::is_timestamp<SourceT>() &&
                                       cudf::is_timestamp<TargetT>())>* = nullptr>
  CUDA_DEVICE_CALLABLE TargetT operator()(SourceT const element)
  {
    // Convert source tick counts into target tick counts without blindly truncating them
    // by dividing the respective duration time periods (which may not work for time before
    // UNIX epoch)
    return TargetT{simt::std::chrono::floor<TargetT::duration>(element.time_since_epoch())};
  }

  template <typename SourceT,
            typename TargetT                                           = _TargetT,
            typename std::enable_if_t<(cudf::is_duration<SourceT>() &&
                                       cudf::is_duration<TargetT>())>* = nullptr>
  CUDA_DEVICE_CALLABLE TargetT operator()(SourceT const element)
  {
    return TargetT{simt::std::chrono::floor<TargetT>(element)};
  }

  template <typename SourceT,
            typename TargetT                                         = _TargetT,
            typename std::enable_if_t<cudf::is_numeric<SourceT>() &&
                                      cudf::is_duration<TargetT>()>* = nullptr>
  CUDA_DEVICE_CALLABLE TargetT operator()(SourceT const element)
  {
    return TargetT{static_cast<typename TargetT::rep>(element)};
  }

  template <typename SourceT,
            typename TargetT                                           = _TargetT,
            typename std::enable_if_t<(cudf::is_timestamp<SourceT>() &&
                                       cudf::is_duration<TargetT>())>* = nullptr>
  CUDA_DEVICE_CALLABLE TargetT operator()(SourceT const element)
  {
    return TargetT{simt::std::chrono::floor<TargetT>(element.time_since_epoch())};
  }

  template <typename SourceT,
            typename TargetT                                        = _TargetT,
            typename std::enable_if_t<cudf::is_duration<SourceT>() &&
                                      cudf::is_numeric<TargetT>()>* = nullptr>
  CUDA_DEVICE_CALLABLE TargetT operator()(SourceT const element)
  {
    return static_cast<TargetT>(element.count());
  }

  template <typename SourceT,
            typename TargetT                                            = _TargetT,
            typename std::enable_if_t<(cudf::is_duration<SourceT>() &&
                                       cudf::is_timestamp<TargetT>())>* = nullptr>
  CUDA_DEVICE_CALLABLE TargetT operator()(SourceT const element)
  {
    return TargetT{simt::std::chrono::floor<TargetT::duration>(element)};
  }
};

template <typename SourceT>
struct dispatch_unary_cast_to {
  column_view input;

  dispatch_unary_cast_to(column_view inp) : input(inp) {}

  // Disallow conversions between timestamps and numeric
  template <
    typename TargetT,
    typename std::enable_if_t<cudf::is_fixed_width<TargetT>() && !cudf::is_fixed_point<TargetT>() &&
                              !(cudf::is_timestamp<SourceT>() && is_numeric<TargetT>()) &&
                              !(cudf::is_timestamp<TargetT>() && is_numeric<SourceT>())>* = nullptr>
  std::unique_ptr<column> operator()(data_type type,
                                     rmm::mr::device_memory_resource* mr,
                                     hipStream_t stream)
  {
    auto size   = input.size();
    auto output = std::make_unique<column>(type,
                                           size,
                                           rmm::device_buffer{size * cudf::size_of(type), 0, mr},
                                           copy_bitmask(input, 0, mr),
                                           input.null_count());

    mutable_column_view output_mutable = *output;

    thrust::transform(rmm::exec_policy(stream)->on(stream),
                      input.begin<SourceT>(),
                      input.end<SourceT>(),
                      output_mutable.begin<TargetT>(),
                      unary_cast<TargetT>{});

    return output;
  }

  template <
    typename TargetT,
    typename std::enable_if_t<!cudf::is_fixed_width<TargetT>() || cudf::is_fixed_point<TargetT>() ||
                              (cudf::is_timestamp<SourceT>() && is_numeric<TargetT>()) ||
                              (cudf::is_timestamp<TargetT>() && is_numeric<SourceT>())>* = nullptr>
  std::unique_ptr<column> operator()(data_type type,
                                     rmm::mr::device_memory_resource* mr,
                                     hipStream_t stream)
  {
    if (!cudf::is_fixed_width<TargetT>())
      CUDF_FAIL("Column type must be numeric or chrono");
    else if (cudf::is_fixed_point<TargetT>())
      CUDF_FAIL("Fixed point unary ops currently not supported");
    else if (cudf::is_timestamp<SourceT>() && is_numeric<TargetT>())
      CUDF_FAIL("Timestamps can be created only from duration");
    else
      CUDF_FAIL("Timestamps cannot be converted to numeric without converting it to a duration");
  }
};

struct dispatch_unary_cast_from {
  column_view input;

  dispatch_unary_cast_from(column_view inp) : input(inp) {}

  template <
    typename T,
    typename std::enable_if_t<cudf::is_fixed_width<T>() && !cudf::is_fixed_point<T>()>* = nullptr>
  std::unique_ptr<column> operator()(data_type type,
                                     rmm::mr::device_memory_resource* mr,
                                     hipStream_t stream)
  {
    return type_dispatcher(type, dispatch_unary_cast_to<T>{input}, type, mr, stream);
  }

  template <typename T, typename std::enable_if_t<cudf::is_fixed_point<T>()>* = nullptr>
  std::unique_ptr<column> operator()(data_type type,
                                     rmm::mr::device_memory_resource* mr,
                                     hipStream_t stream)
  {
    CUDF_FAIL("Fixed point unary ops not supported yet");
  }

  template <typename T, typename std::enable_if_t<!cudf::is_fixed_width<T>()>* = nullptr>
  std::unique_ptr<column> operator()(data_type type,
                                     rmm::mr::device_memory_resource* mr,
                                     hipStream_t stream)
  {
    CUDF_FAIL("Column type must be numeric or chrono");
  }
};

std::unique_ptr<column> cast(column_view const& input,
                             data_type type,
                             rmm::mr::device_memory_resource* mr,
                             hipStream_t stream)
{
  CUDF_EXPECTS(is_fixed_width(type), "Unary cast type must be fixed-width.");

  return type_dispatcher(input.type(), detail::dispatch_unary_cast_from{input}, type, mr, stream);
}

}  // namespace detail

std::unique_ptr<column> cast(column_view const& input,
                             data_type type,
                             rmm::mr::device_memory_resource* mr)
{
  CUDF_FUNC_RANGE();
  return detail::cast(input, type, mr);
}

}  // namespace cudf
