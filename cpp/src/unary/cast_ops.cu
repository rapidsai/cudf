#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <cudf/column/column.hpp>
#include <cudf/detail/nvtx/ranges.hpp>
#include <cudf/detail/unary.hpp>
#include <cudf/fixed_point/fixed_point.hpp>
#include <cudf/null_mask.hpp>
#include <cudf/unary.hpp>
#include <cudf/utilities/traits.hpp>

#include <rmm/thrust_rmm_allocator.h>

namespace cudf {
namespace detail {
template <typename _TargetT>
struct unary_cast {
  template <typename SourceT,
            typename TargetT                                          = _TargetT,
            typename std::enable_if_t<(cudf::is_numeric<SourceT>() &&
                                       cudf::is_numeric<TargetT>())>* = nullptr>
  CUDA_DEVICE_CALLABLE TargetT operator()(SourceT const element)
  {
    return static_cast<TargetT>(element);
  }

  template <typename SourceT,
            typename TargetT                                            = _TargetT,
            typename std::enable_if_t<(cudf::is_timestamp<SourceT>() &&
                                       cudf::is_timestamp<TargetT>())>* = nullptr>
  CUDA_DEVICE_CALLABLE TargetT operator()(SourceT const element)
  {
    // Convert source tick counts into target tick counts without blindly truncating them
    // by dividing the respective duration time periods (which may not work for time before
    // UNIX epoch)
    return TargetT{simt::std::chrono::floor<TargetT::duration>(element.time_since_epoch())};
  }

  template <typename SourceT,
            typename TargetT                                           = _TargetT,
            typename std::enable_if_t<(cudf::is_duration<SourceT>() &&
                                       cudf::is_duration<TargetT>())>* = nullptr>
  CUDA_DEVICE_CALLABLE TargetT operator()(SourceT const element)
  {
    return TargetT{simt::std::chrono::floor<TargetT>(element)};
  }

  template <typename SourceT,
            typename TargetT                                         = _TargetT,
            typename std::enable_if_t<cudf::is_numeric<SourceT>() &&
                                      cudf::is_duration<TargetT>()>* = nullptr>
  CUDA_DEVICE_CALLABLE TargetT operator()(SourceT const element)
  {
    return TargetT{static_cast<typename TargetT::rep>(element)};
  }

  template <typename SourceT,
            typename TargetT                                           = _TargetT,
            typename std::enable_if_t<(cudf::is_timestamp<SourceT>() &&
                                       cudf::is_duration<TargetT>())>* = nullptr>
  CUDA_DEVICE_CALLABLE TargetT operator()(SourceT const element)
  {
    return TargetT{simt::std::chrono::floor<TargetT>(element.time_since_epoch())};
  }

  template <typename SourceT,
            typename TargetT                                        = _TargetT,
            typename std::enable_if_t<cudf::is_duration<SourceT>() &&
                                      cudf::is_numeric<TargetT>()>* = nullptr>
  CUDA_DEVICE_CALLABLE TargetT operator()(SourceT const element)
  {
    return static_cast<TargetT>(element.count());
  }

  template <typename SourceT,
            typename TargetT                                            = _TargetT,
            typename std::enable_if_t<(cudf::is_duration<SourceT>() &&
                                       cudf::is_timestamp<TargetT>())>* = nullptr>
  CUDA_DEVICE_CALLABLE TargetT operator()(SourceT const element)
  {
    return TargetT{simt::std::chrono::floor<TargetT::duration>(element)};
  }
};

template <typename _SourceT, typename _TargetT>
struct fixed_point_unary_cast {
  numeric::scale_type scale;
  using FixedPointT = std::conditional_t<cudf::is_fixed_point<_SourceT>(), _SourceT, _TargetT>;
  using DeviceT     = device_storage_type_t<FixedPointT>;

  template <typename SourceT                                          = _SourceT,
            typename TargetT                                          = _TargetT,
            typename std::enable_if_t<(cudf::is_fixed_point<_SourceT>() &&
                                       cudf::is_numeric<TargetT>())>* = nullptr>
  CUDA_DEVICE_CALLABLE TargetT operator()(DeviceT const element)
  {
    auto const fp = SourceT{numeric::scaled_integer<DeviceT>{element, scale}};
    return static_cast<TargetT>(fp);
  }

  template <typename SourceT                                              = _SourceT,
            typename TargetT                                              = _TargetT,
            typename std::enable_if_t<(cudf::is_numeric<_SourceT>() &&
                                       cudf::is_fixed_point<TargetT>())>* = nullptr>
  CUDA_DEVICE_CALLABLE DeviceT operator()(SourceT const element)
  {
    auto const fp = TargetT{element, scale};
    return numeric::scaled_integer<DeviceT>{fp}.value;
  }
};

template <typename From, typename To>
constexpr inline auto is_supported_non_fixed_point_cast()
{
  return cudf::is_fixed_width<To>() &&
         // Disallow fixed_point here (requires different specialization)
         !(cudf::is_fixed_point<From>() || cudf::is_fixed_point<To>()) &&
         // Disallow conversions between timestamps and numeric
         !(cudf::is_timestamp<From>() && is_numeric<To>()) &&
         !(cudf::is_timestamp<To>() && is_numeric<From>());
}

template <typename From, typename To>
constexpr inline auto is_supported_fixed_point_cast()
{
  return (cudf::is_fixed_point<From>() && cudf::is_numeric<To>()) ||
         (cudf::is_numeric<From>() && cudf::is_fixed_point<To>());
}

template <typename From, typename To>
constexpr inline auto is_supported_cast()
{
  return is_supported_non_fixed_point_cast<From, To>() || is_supported_fixed_point_cast<From, To>();
}

template <typename SourceT>
struct dispatch_unary_cast_to {
  column_view input;

  dispatch_unary_cast_to(column_view inp) : input(inp) {}

  template <
    typename TargetT,
    typename std::enable_if_t<is_supported_non_fixed_point_cast<SourceT, TargetT>()>* = nullptr>
  std::unique_ptr<column> operator()(data_type type,
                                     rmm::mr::device_memory_resource* mr,
                                     hipStream_t stream)
  {
    auto const size = input.size();
    auto output =
      std::make_unique<column>(type,
                               size,
                               rmm::device_buffer{size * cudf::size_of(type), stream, mr},
                               copy_bitmask(input, stream, mr),
                               input.null_count());

    mutable_column_view output_mutable = *output;

    thrust::transform(rmm::exec_policy(stream)->on(stream),
                      input.begin<SourceT>(),
                      input.end<SourceT>(),
                      output_mutable.begin<TargetT>(),
                      unary_cast<TargetT>{});

    return output;
  }

  template <typename TargetT,
            typename std::enable_if_t<cudf::is_fixed_point<SourceT>() &&
                                      cudf::is_numeric<TargetT>()>* = nullptr>
  std::unique_ptr<column> operator()(data_type type,
                                     rmm::mr::device_memory_resource* mr,
                                     hipStream_t stream)
  {
    auto const size = input.size();
    auto output =
      std::make_unique<column>(type,
                               size,
                               rmm::device_buffer{size * cudf::size_of(type), stream, mr},
                               copy_bitmask(input, stream, mr),
                               input.null_count());

    mutable_column_view output_mutable = *output;

    using DeviceT    = device_storage_type_t<SourceT>;
    auto const scale = numeric::scale_type{input.type().scale()};

    thrust::transform(rmm::exec_policy(stream)->on(stream),
                      input.begin<DeviceT>(),
                      input.end<DeviceT>(),
                      output_mutable.begin<TargetT>(),
                      fixed_point_unary_cast<SourceT, TargetT>{scale});

    return output;
  }

  template <typename TargetT,
            typename std::enable_if_t<cudf::is_numeric<SourceT>() &&
                                      cudf::is_fixed_point<TargetT>()>* = nullptr>
  std::unique_ptr<column> operator()(data_type type,
                                     rmm::mr::device_memory_resource* mr,
                                     hipStream_t stream)
  {
    auto const size = input.size();
    auto output =
      std::make_unique<column>(type,
                               size,
                               rmm::device_buffer{size * cudf::size_of(type), stream, mr},
                               copy_bitmask(input, stream, mr),
                               input.null_count());

    mutable_column_view output_mutable = *output;

    using DeviceT    = device_storage_type_t<TargetT>;
    auto const scale = numeric::scale_type{type.scale()};

    thrust::transform(rmm::exec_policy(stream)->on(stream),
                      input.begin<SourceT>(),
                      input.end<SourceT>(),
                      output_mutable.begin<DeviceT>(),
                      fixed_point_unary_cast<SourceT, TargetT>{scale});

    return output;
  }

  template <typename TargetT,
            typename std::enable_if_t<not is_supported_cast<SourceT, TargetT>()>* = nullptr>
  std::unique_ptr<column> operator()(data_type type,
                                     rmm::mr::device_memory_resource* mr,
                                     hipStream_t stream)
  {
    if (!cudf::is_fixed_width<TargetT>())
      CUDF_FAIL("Column type must be numeric or chrono or decimal32/64");
    else if (cudf::is_fixed_point<SourceT>())
      CUDF_FAIL("Currently only decimal32/64 to floating point/integral is supported");
    else if (cudf::is_timestamp<SourceT>() && is_numeric<TargetT>())
      CUDF_FAIL("Timestamps can be created only from duration");
    else
      CUDF_FAIL("Timestamps cannot be converted to numeric without converting it to a duration");
  }
};

struct dispatch_unary_cast_from {
  column_view input;

  dispatch_unary_cast_from(column_view inp) : input(inp) {}

  template <typename T, typename std::enable_if_t<cudf::is_fixed_width<T>()>* = nullptr>
  std::unique_ptr<column> operator()(data_type type,
                                     rmm::mr::device_memory_resource* mr,
                                     hipStream_t stream)
  {
    return type_dispatcher(type, dispatch_unary_cast_to<T>{input}, type, mr, stream);
  }

  template <typename T, typename std::enable_if_t<!cudf::is_fixed_width<T>()>* = nullptr>
  std::unique_ptr<column> operator()(data_type type,
                                     rmm::mr::device_memory_resource* mr,
                                     hipStream_t stream)
  {
    CUDF_FAIL("Column type must be numeric or chrono or decimal32/64");
  }
};

std::unique_ptr<column> cast(column_view const& input,
                             data_type type,
                             rmm::mr::device_memory_resource* mr,
                             hipStream_t stream)
{
  CUDF_EXPECTS(is_fixed_width(type), "Unary cast type must be fixed-width.");

  return type_dispatcher(input.type(), detail::dispatch_unary_cast_from{input}, type, mr, stream);
}

}  // namespace detail

std::unique_ptr<column> cast(column_view const& input,
                             data_type type,
                             rmm::mr::device_memory_resource* mr)
{
  CUDF_FUNC_RANGE();
  return detail::cast(input, type, mr);
}

}  // namespace cudf
