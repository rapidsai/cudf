#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019-2020, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <cudf/aggregation.hpp>
#include <cudf/column/column_device_view.cuh>
#include <cudf/column/column_factories.hpp>
#include <cudf/column/column_view.hpp>
#include <cudf/copying.hpp>
#include <cudf/detail/aggregation/aggregation.cuh>
#include <cudf/detail/copy.hpp>
#include <cudf/detail/gather.hpp>
#include <cudf/detail/groupby/sort_helper.hpp>
#include <cudf/detail/nvtx/ranges.hpp>
#include <cudf/detail/utilities/cuda.cuh>
#include <cudf/rolling.hpp>
#include <cudf/types.hpp>
#include <cudf/utilities/bit.hpp>
#include <rolling/rolling_detail.hpp>
#include <rolling/rolling_jit_detail.hpp>

#include <jit/launcher.h>
#include <jit/parser.h>
#include <jit/type.h>
#include <rolling/jit/code/code.h>

#include <jit/bit.hpp.jit>
#include <jit/rolling_jit_detail.hpp.jit>
#include <jit/types.hpp.jit>

#include <thrust/binary_search.h>
#include <rmm/device_scalar.hpp>

#include <rmm/thrust_rmm_allocator.h>
#include <thrust/detail/execution_policy.h>
#include <thrust/execution_policy.h>
#include <thrust/find.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/transform.h>
#include <cudf/detail/aggregation/aggregation.hpp>
#include <cudf/detail/utilities/device_operators.cuh>
#include <cudf/utilities/error.hpp>
#include <cudf/utilities/traits.hpp>
#include <memory>

namespace cudf {
namespace detail {
namespace {  // anonymous
/**
 * @brief Only COUNT_VALID operation is executed and count is updated
 *        depending on `min_periods` and returns true if it was
 *        valid, else false.
 */
template <typename InputType,
          typename OutputType,
          typename agg_op,
          aggregation::Kind op,
          bool has_nulls,
          std::enable_if_t<op == aggregation::COUNT_VALID>* = nullptr>
bool __device__ process_rolling_window(column_device_view input,
                                       column_device_view ignored_default_outputs,
                                       mutable_column_device_view output,
                                       size_type start_index,
                                       size_type end_index,
                                       size_type current_index,
                                       size_type min_periods)
{
  // declare this as volatile to avoid some compiler optimizations that lead to incorrect results
  // for CUDA 10.0 and below (fixed in CUDA 10.1)
  volatile cudf::size_type count = 0;

  bool output_is_valid = ((end_index - start_index) >= min_periods);

  if (output_is_valid) {
    if (!has_nulls) {
      count = end_index - start_index;
    } else {
      count = thrust::count_if(thrust::seq,
                               thrust::make_counting_iterator(start_index),
                               thrust::make_counting_iterator(end_index),
                               [&input](auto i) { return input.is_valid_nocheck(i); });
    }
    output.element<OutputType>(current_index) = count;
  }

  return output_is_valid;
}

/**
 * @brief Only COUNT_ALL operation is executed and count is updated
 *        depending on `min_periods` and returns true if it was
 *        valid, else false.
 */
template <typename InputType,
          typename OutputType,
          typename agg_op,
          aggregation::Kind op,
          bool has_nulls,
          std::enable_if_t<op == aggregation::COUNT_ALL>* = nullptr>
bool __device__ process_rolling_window(column_device_view input,
                                       column_device_view ignored_default_outputs,
                                       mutable_column_device_view output,
                                       size_type start_index,
                                       size_type end_index,
                                       size_type current_index,
                                       size_type min_periods)
{
  cudf::size_type count = end_index - start_index;

  bool output_is_valid                      = (count >= min_periods);
  output.element<OutputType>(current_index) = count;

  return output_is_valid;
}

/**
 * @brief Calculates row-number within [start_index, end_index).
 *        Count is updated depending on `min_periods`
 *        Returns true if it was valid, else false.
 */
template <typename InputType,
          typename OutputType,
          typename agg_op,
          aggregation::Kind op,
          bool has_nulls,
          std::enable_if_t<op == aggregation::ROW_NUMBER>* = nullptr>
bool __device__ process_rolling_window(column_device_view input,
                                       column_device_view ignored_default_outputs,
                                       mutable_column_device_view output,
                                       size_type start_index,
                                       size_type end_index,
                                       size_type current_index,
                                       size_type min_periods)
{
  bool output_is_valid                      = ((end_index - start_index) >= min_periods);
  output.element<OutputType>(current_index) = ((current_index - start_index) + 1);

  return output_is_valid;
}

/**
 * @brief LEAD(N): Returns the row from the input column, at the specified offset past the
 *        current row.
 * If the offset crosses the grouping boundary or column boundary for
 * a given row, a "default" value is returned. The "default" value is null, by default.
 *
 * E.g. Consider an input column with the following values and grouping:
 *      [10, 11, 12, 13,   20, 21, 22, 23]
 *      <------G1----->   <------G2------>
 *
 * LEAD(input_col, 1) yields:
 *      [11, 12, 13, null,  21, 22, 23, null]
 *
 * LEAD(input_col, 1, 99) (where 99 indicates the default) yields:
 *      [11, 12, 13, 99,  21, 22, 23, 99]
 */
template <typename InputType,
          typename OutputType,
          typename agg_op,
          aggregation::Kind op,
          bool has_nulls>
std::enable_if_t<(op == aggregation::LEAD) && (cudf::is_fixed_width<InputType>()), bool> __device__
process_rolling_window(column_device_view input,
                       column_device_view default_outputs,
                       mutable_column_device_view output,
                       size_type start_index,
                       size_type end_index,
                       size_type current_index,
                       size_type min_periods,
                       agg_op device_agg_op)
{
  // Offsets have already been normalized.
  auto row_offset = device_agg_op.row_offset;

  // Check if row is invalid.
  if (row_offset > (end_index - current_index - 1)) {
    // Invalid row marked. Use default value, if available.
    if (default_outputs.size() == 0 || default_outputs.is_null(current_index)) { return false; }

    output.element<OutputType>(current_index) = default_outputs.element<OutputType>(current_index);
    return true;
  }

  // Not an invalid row.
  auto index   = current_index + row_offset;
  auto is_null = input.is_null(index);
  if (!is_null) { output.element<OutputType>(current_index) = input.element<InputType>(index); }
  return !is_null;
}

/**
 * @brief LAG(N): returns the row from the input column at the specified offset preceding
 *        the current row.
 * If the offset crosses the grouping boundary or column boundary for
 * a given row, a "default" value is returned. The "default" value is null, by default.
 *
 * E.g. Consider an input column with the following values and grouping:
 *      [10, 11, 12, 13,   20, 21, 22, 23]
 *      <------G1----->   <------G2------>
 *
 * LAG(input_col, 2) yields:
 *      [null, null, 10, 11, null, null, 20, 21]
 * LAG(input_col, 2, 99) yields:
 *      [99, 99, 10, 11, 99, 99, 20, 21]
 */
template <typename InputType,
          typename OutputType,
          typename agg_op,
          aggregation::Kind op,
          bool has_nulls>
std::enable_if_t<(op == aggregation::LAG) && (cudf::is_fixed_width<InputType>()), bool> __device__
process_rolling_window(column_device_view input,
                       column_device_view default_outputs,
                       mutable_column_device_view output,
                       size_type start_index,
                       size_type end_index,
                       size_type current_index,
                       size_type min_periods,
                       agg_op device_agg_op)
{
  // Offsets have already been normalized.
  auto row_offset = device_agg_op.row_offset;

  // Check if row is invalid.
  if (row_offset > (current_index - start_index)) {
    // Invalid row marked. Use default value, if available.
    if (default_outputs.size() == 0 || default_outputs.is_null(current_index)) { return false; }

    output.element<OutputType>(current_index) = default_outputs.element<OutputType>(current_index);
    return true;
  }

  // Not an invalid row.
  auto index   = current_index - row_offset;
  auto is_null = input.is_null(index);
  if (!is_null) { output.element<OutputType>(current_index) = input.element<InputType>(index); }
  return !is_null;
}

/**
 * @brief Only used for `string_view` type to get ARGMIN and ARGMAX, which
 *        will be used to gather MIN and MAX. And returns true if the
 *        operation was valid, else false.
 */
template <typename InputType,
          typename OutputType,
          typename agg_op,
          aggregation::Kind op,
          bool has_nulls,
          std::enable_if_t<(op == aggregation::ARGMIN or op == aggregation::ARGMAX) and
                           std::is_same<InputType, cudf::string_view>::value>* = nullptr>
bool __device__ process_rolling_window(column_device_view input,
                                       column_device_view ignored_default_outputs,
                                       mutable_column_device_view output,
                                       size_type start_index,
                                       size_type end_index,
                                       size_type current_index,
                                       size_type min_periods)
{
  // declare this as volatile to avoid some compiler optimizations that lead to incorrect results
  // for CUDA 10.0 and below (fixed in CUDA 10.1)
  volatile cudf::size_type count = 0;
  InputType val                  = agg_op::template identity<InputType>();
  OutputType val_index           = (op == aggregation::ARGMIN) ? ARGMIN_SENTINEL : ARGMAX_SENTINEL;

  for (size_type j = start_index; j < end_index; j++) {
    if (!has_nulls || input.is_valid(j)) {
      InputType element = input.element<InputType>(j);
      val               = agg_op{}(element, val);
      if (val == element) { val_index = j; }
      count++;
    }
  }

  bool output_is_valid = (count >= min_periods);
  // -1 will help identify null elements while gathering for Min and Max
  // In case of count, this would be null, so doesn't matter.
  output.element<OutputType>(current_index) = (output_is_valid) ? val_index : -1;

  // The gather mask shouldn't contain null values, so
  // always return zero
  return true;
}

/**
 * @brief Operates on only fixed-width types and returns true if the
 *        operation was valid, else false.
 */
template <typename InputType,
          typename OutputType,
          typename agg_op,
          aggregation::Kind op,
          bool has_nulls,
          std::enable_if_t<!std::is_same<InputType, cudf::string_view>::value and
                           !(op == aggregation::COUNT_VALID || op == aggregation::COUNT_ALL ||
                             op == aggregation::ROW_NUMBER || op == aggregation::LEAD ||
                             op == aggregation::LAG)>* = nullptr>
bool __device__ process_rolling_window(column_device_view input,
                                       column_device_view ignored_default_outputs,
                                       mutable_column_device_view output,
                                       size_type start_index,
                                       size_type end_index,
                                       size_type current_index,
                                       size_type min_periods)
{
  // declare this as volatile to avoid some compiler optimizations that lead to incorrect results
  // for CUDA 10.0 and below (fixed in CUDA 10.1)
  volatile cudf::size_type count = 0;
  OutputType val                 = agg_op::template identity<OutputType>();

  for (size_type j = start_index; j < end_index; j++) {
    if (!has_nulls || input.is_valid(j)) {
      OutputType element = input.element<InputType>(j);
      val                = agg_op{}(element, val);
      count++;
    }
  }

  bool output_is_valid = (count >= min_periods);

  // store the output value, one per thread
  cudf::detail::rolling_store_output_functor<OutputType, op == aggregation::MEAN>{}(
    output.element<OutputType>(current_index), val, count);

  return output_is_valid;
}

/**
 * @brief Computes the rolling window function
 *
 * @tparam InputType  Datatype of `input`
 * @tparam OutputType  Datatype of `output`
 * @tparam agg_op  A functor that defines the aggregation operation
 * @tparam op The aggregation operator (enum value)
 * @tparam block_size CUDA block size for the kernel
 * @tparam has_nulls true if the input column has nulls
 * @tparam PrecedingWindowIterator iterator type (inferred)
 * @tparam FollowingWindowIterator iterator type (inferred)
 * @param input Input column device view
 * @param output Output column device view
 * @param preceding_window_begin[in] Rolling window size iterator, accumulates from
 *                in_col[i-preceding_window] to in_col[i] inclusive
 * @param following_window_begin[in] Rolling window size iterator in the forward
 *                direction, accumulates from in_col[i] to
 *                in_col[i+following_window] inclusive
 * @param min_periods[in]  Minimum number of observations in window required to
 *                have a value, otherwise 0 is stored in the valid bit mask
 */
template <typename InputType,
          typename OutputType,
          typename agg_op,
          aggregation::Kind op,
          int block_size,
          bool has_nulls,
          typename PrecedingWindowIterator,
          typename FollowingWindowIterator>
__launch_bounds__(block_size) __global__
  void gpu_rolling(column_device_view input,
                   column_device_view default_outputs,
                   mutable_column_device_view output,
                   size_type* __restrict__ output_valid_count,
                   PrecedingWindowIterator preceding_window_begin,
                   FollowingWindowIterator following_window_begin,
                   size_type min_periods)
{
  size_type i      = blockIdx.x * block_size + threadIdx.x;
  size_type stride = block_size * gridDim.x;

  size_type warp_valid_count{0};

  auto active_threads = __ballot_sync(0xffffffff, i < input.size());
  while (i < input.size()) {
    size_type preceding_window = preceding_window_begin[i];
    size_type following_window = following_window_begin[i];

    // compute bounds
    size_type start       = min(input.size(), max(0, i - preceding_window + 1));
    size_type end         = min(input.size(), max(0, i + following_window + 1));
    size_type start_index = min(start, end);
    size_type end_index   = max(start, end);

    // aggregate
    // TODO: We should explore using shared memory to avoid redundant loads.
    //       This might require separating the kernel into a special version
    //       for dynamic and static sizes.

    volatile bool output_is_valid = false;
    output_is_valid = process_rolling_window<InputType, OutputType, agg_op, op, has_nulls>(
      input, default_outputs, output, start_index, end_index, i, min_periods);

    // set the mask
    cudf::bitmask_type result_mask{__ballot_sync(active_threads, output_is_valid)};

    // only one thread writes the mask
    if (0 == threadIdx.x % cudf::detail::warp_size) {
      output.set_mask_word(cudf::word_index(i), result_mask);
      warp_valid_count += __popc(result_mask);
    }

    // process next element
    i += stride;
    active_threads = __ballot_sync(active_threads, i < input.size());
  }

  // sum the valid counts across the whole block
  size_type block_valid_count =
    cudf::detail::single_lane_block_sum_reduce<block_size, 0>(warp_valid_count);

  if (threadIdx.x == 0) { atomicAdd(output_valid_count, block_valid_count); }
}

template <typename InputType,
          typename OutputType,
          typename agg_op,
          aggregation::Kind op,
          int block_size,
          bool has_nulls,
          typename PrecedingWindowIterator,
          typename FollowingWindowIterator>
__launch_bounds__(block_size) __global__
  void gpu_rolling(column_device_view input,
                   column_device_view default_outputs,
                   mutable_column_device_view output,
                   size_type* __restrict__ output_valid_count,
                   PrecedingWindowIterator preceding_window_begin,
                   FollowingWindowIterator following_window_begin,
                   size_type min_periods,
                   agg_op device_agg_op)
{
  size_type i      = blockIdx.x * block_size + threadIdx.x;
  size_type stride = block_size * gridDim.x;

  size_type warp_valid_count{0};

  auto active_threads = __ballot_sync(0xffffffff, i < input.size());
  while (i < input.size()) {
    size_type preceding_window = preceding_window_begin[i];
    size_type following_window = following_window_begin[i];

    // compute bounds
    size_type start       = min(input.size(), max(0, i - preceding_window + 1));
    size_type end         = min(input.size(), max(0, i + following_window + 1));
    size_type start_index = min(start, end);
    size_type end_index   = max(start, end);

    // aggregate
    // TODO: We should explore using shared memory to avoid redundant loads.
    //       This might require separating the kernel into a special version
    //       for dynamic and static sizes.

    volatile bool output_is_valid = false;
    output_is_valid = process_rolling_window<InputType, OutputType, agg_op, op, has_nulls>(
      input, default_outputs, output, start_index, end_index, i, min_periods, device_agg_op);

    // set the mask
    cudf::bitmask_type result_mask{__ballot_sync(active_threads, output_is_valid)};

    // only one thread writes the mask
    if (0 == threadIdx.x % cudf::detail::warp_size) {
      output.set_mask_word(cudf::word_index(i), result_mask);
      warp_valid_count += __popc(result_mask);
    }

    // process next element
    i += stride;
    active_threads = __ballot_sync(active_threads, i < input.size());
  }

  // sum the valid counts across the whole block
  size_type block_valid_count =
    cudf::detail::single_lane_block_sum_reduce<block_size, 0>(warp_valid_count);

  if (threadIdx.x == 0) { atomicAdd(output_valid_count, block_valid_count); }
}

template <typename InputType>
struct rolling_window_launcher {
  template <typename T,
            typename agg_op,
            aggregation::Kind op,
            typename PrecedingWindowIterator,
            typename FollowingWindowIterator>
  size_type kernel_launcher(column_view const& input,
                            column_view const& default_outputs,
                            mutable_column_view& output,
                            PrecedingWindowIterator preceding_window_begin,
                            FollowingWindowIterator following_window_begin,
                            size_type min_periods,
                            std::unique_ptr<aggregation> const& agg,
                            hipStream_t stream)
  {
    constexpr cudf::size_type block_size = 256;
    cudf::detail::grid_1d grid(input.size(), block_size);

    auto input_device_view           = column_device_view::create(input, stream);
    auto output_device_view          = mutable_column_device_view::create(output, stream);
    auto default_outputs_device_view = column_device_view::create(default_outputs, stream);

    rmm::device_scalar<size_type> device_valid_count{0, stream};

    if (input.has_nulls()) {
      gpu_rolling<T, target_type_t<InputType, op>, agg_op, op, block_size, true>
        <<<grid.num_blocks, block_size, 0, stream>>>(*input_device_view,
                                                     *default_outputs_device_view,
                                                     *output_device_view,
                                                     device_valid_count.data(),
                                                     preceding_window_begin,
                                                     following_window_begin,
                                                     min_periods);
    } else {
      gpu_rolling<T, target_type_t<InputType, op>, agg_op, op, block_size, false>
        <<<grid.num_blocks, block_size, 0, stream>>>(*input_device_view,
                                                     *default_outputs_device_view,
                                                     *output_device_view,
                                                     device_valid_count.data(),
                                                     preceding_window_begin,
                                                     following_window_begin,
                                                     min_periods);
    }

    size_type valid_count = device_valid_count.value(stream);

    // check the stream for debugging
    CHECK_CUDA(stream);

    return valid_count;
  }

  template <typename T,
            typename agg_op,
            aggregation::Kind op,
            typename PrecedingWindowIterator,
            typename FollowingWindowIterator>
  size_type kernel_launcher(column_view const& input,
                            column_view const& default_outputs,
                            mutable_column_view& output,
                            PrecedingWindowIterator preceding_window_begin,
                            FollowingWindowIterator following_window_begin,
                            size_type min_periods,
                            std::unique_ptr<aggregation> const& agg,
                            agg_op const& device_agg_op,
                            hipStream_t stream)
  {
    constexpr cudf::size_type block_size = 256;
    cudf::detail::grid_1d grid(input.size(), block_size);

    auto input_device_view           = column_device_view::create(input, stream);
    auto output_device_view          = mutable_column_device_view::create(output, stream);
    auto default_outputs_device_view = column_device_view::create(default_outputs, stream);

    rmm::device_scalar<size_type> device_valid_count{0, stream};

    if (input.has_nulls()) {
      gpu_rolling<T, target_type_t<InputType, op>, agg_op, op, block_size, true>
        <<<grid.num_blocks, block_size, 0, stream>>>(*input_device_view,
                                                     *default_outputs_device_view,
                                                     *output_device_view,
                                                     device_valid_count.data(),
                                                     preceding_window_begin,
                                                     following_window_begin,
                                                     min_periods,
                                                     device_agg_op);
    } else {
      gpu_rolling<T, target_type_t<InputType, op>, agg_op, op, block_size, false>
        <<<grid.num_blocks, block_size, 0, stream>>>(*input_device_view,
                                                     *default_outputs_device_view,
                                                     *output_device_view,
                                                     device_valid_count.data(),
                                                     preceding_window_begin,
                                                     following_window_begin,
                                                     min_periods,
                                                     device_agg_op);
    }

    size_type valid_count = device_valid_count.value(stream);

    // check the stream for debugging
    CHECK_CUDA(stream);

    return valid_count;
  }

  // This launch is only for fixed width columns with valid aggregation option
  // numeric: All
  // timestamp: MIN, MAX, COUNT_VALID, COUNT_ALL, ROW_NUMBER
  // string, dictionary, list : COUNT_VALID, COUNT_ALL, ROW_NUMBER
  template <typename T,
            typename agg_op,
            aggregation::Kind op,
            typename PrecedingWindowIterator,
            typename FollowingWindowIterator>
  std::enable_if_t<cudf::detail::is_rolling_supported<T, agg_op, op>() and
                     !cudf::detail::is_rolling_string_specialization<T, agg_op, op>(),
                   std::unique_ptr<column>>
  launch(column_view const& input,
         column_view const& default_outputs,
         PrecedingWindowIterator preceding_window_begin,
         FollowingWindowIterator following_window_begin,
         size_type min_periods,
         std::unique_ptr<aggregation> const& agg,
         rmm::mr::device_memory_resource* mr,
         hipStream_t stream)
  {
    if (input.is_empty()) return empty_like(input);

    auto output = make_fixed_width_column(
      target_type(input.type(), op), input.size(), mask_state::UNINITIALIZED, stream, mr);

    cudf::mutable_column_view output_view = output->mutable_view();
    auto valid_count =
      kernel_launcher<T, agg_op, op, PrecedingWindowIterator, FollowingWindowIterator>(
        input,
        default_outputs,
        output_view,
        preceding_window_begin,
        following_window_begin,
        min_periods,
        agg,
        stream);

    output->set_null_count(output->size() - valid_count);

    return output;
  }

  // This launch is only for string specializations
  // string: MIN, MAX
  template <typename T,
            typename agg_op,
            aggregation::Kind op,
            typename PrecedingWindowIterator,
            typename FollowingWindowIterator>
  std::enable_if_t<cudf::detail::is_rolling_string_specialization<T, agg_op, op>(),
                   std::unique_ptr<column>>
  launch(column_view const& input,
         column_view const& default_outputs,
         PrecedingWindowIterator preceding_window_begin,
         FollowingWindowIterator following_window_begin,
         size_type min_periods,
         std::unique_ptr<aggregation> const& agg,
         rmm::mr::device_memory_resource* mr,
         hipStream_t stream)
  {
    if (input.is_empty()) return empty_like(input);

    auto output = make_numeric_column(cudf::data_type{cudf::type_to_id<size_type>()},
                                      input.size(),
                                      cudf::mask_state::UNINITIALIZED,
                                      stream,
                                      mr);

    cudf::mutable_column_view output_view = output->mutable_view();

    // Passing the agg_op and aggregation::Kind as constant to group them in pair, else it
    // evolves to error when try to use agg_op as compiler tries different combinations
    if (op == aggregation::MIN) {
      kernel_launcher<T,
                      DeviceMin,
                      aggregation::ARGMIN,
                      PrecedingWindowIterator,
                      FollowingWindowIterator>(input,
                                               default_outputs,
                                               output_view,
                                               preceding_window_begin,
                                               following_window_begin,
                                               min_periods,
                                               agg,
                                               stream);
    } else if (op == aggregation::MAX) {
      kernel_launcher<T,
                      DeviceMax,
                      aggregation::ARGMAX,
                      PrecedingWindowIterator,
                      FollowingWindowIterator>(input,
                                               default_outputs,
                                               output_view,
                                               preceding_window_begin,
                                               following_window_begin,
                                               min_periods,
                                               agg,
                                               stream);
    } else {
      CUDF_FAIL("MIN and MAX are the only supported aggregation types for string columns");
    }

    // The rows that represent null elements will be having negative values in gather map,
    // and that's why nullify_out_of_bounds/ignore_out_of_bounds is true.
    auto output_table = detail::gather(table_view{{input}},
                                       output->view(),
                                       detail::out_of_bounds_policy::IGNORE,
                                       detail::negative_index_policy::NOT_ALLOWED,
                                       stream,
                                       mr);
    return std::make_unique<cudf::column>(std::move(output_table->get_column(0)));
  }

  // Deals with invalid column and/or aggregation options
  template <typename T,
            typename agg_op,
            aggregation::Kind op,
            typename PrecedingWindowIterator,
            typename FollowingWindowIterator>
  std::enable_if_t<!cudf::detail::is_rolling_supported<T, agg_op, op>() and
                     !cudf::detail::is_rolling_string_specialization<T, agg_op, op>(),
                   std::unique_ptr<column>>
  launch(column_view const& input,
         column_view const& default_outputs,
         PrecedingWindowIterator preceding_window_begin,
         FollowingWindowIterator following_window_begin,
         size_type min_periods,
         std::unique_ptr<aggregation> const& agg,
         rmm::mr::device_memory_resource* mr,
         hipStream_t stream)
  {
    CUDF_FAIL("Aggregation operator and/or input type combination is invalid");
  }

  template <typename T,
            typename agg_op,
            aggregation::Kind op,
            typename PrecedingWindowIterator,
            typename FollowingWindowIterator>
  std::enable_if_t<cudf::is_fixed_width<T>() and
                     (op == aggregation::LEAD || op == aggregation::LAG),
                   std::unique_ptr<column>>
  launch(column_view const& input,
         column_view const& default_outputs,
         PrecedingWindowIterator preceding_window_begin,
         FollowingWindowIterator following_window_begin,
         size_type min_periods,
         std::unique_ptr<aggregation> const& agg,
         agg_op const& device_agg_op,
         rmm::mr::device_memory_resource* mr,
         hipStream_t stream)
  {
    if (input.is_empty()) return empty_like(input);

    CUDF_EXPECTS(default_outputs.type().id() == input.type().id(),
                 "Defaults column type must match input column.");  // Because LEAD/LAG.

    // For LEAD(0)/LAG(0), no computation need be performed.
    // Return copy of input.
    if (0 == static_cast<cudf::detail::lead_lag_aggregation*>(agg.get())->row_offset) {
      return std::make_unique<column>(input, stream, mr);
    }

    auto output = make_fixed_width_column(
      target_type(input.type(), op), input.size(), mask_state::UNINITIALIZED, stream, mr);

    cudf::mutable_column_view output_view = output->mutable_view();
    auto valid_count =
      kernel_launcher<T, agg_op, op, PrecedingWindowIterator, FollowingWindowIterator>(
        input,
        default_outputs,
        output_view,
        preceding_window_begin,
        following_window_begin,
        min_periods,
        agg,
        device_agg_op,
        stream);

    output->set_null_count(output->size() - valid_count);

    return output;
  }

  // Deals with invalid column and/or aggregation options
  template <typename T,
            typename agg_op,
            aggregation::Kind op,
            typename PrecedingWindowIterator,
            typename FollowingWindowIterator>
  std::enable_if_t<!(op == aggregation::LEAD || op == aggregation::LAG) ||
                     !cudf::is_fixed_width<T>(),
                   std::unique_ptr<column>>
  launch(column_view const& input,
         column_view const& default_outputs,
         PrecedingWindowIterator preceding_window_begin,
         FollowingWindowIterator following_window_begin,
         size_type min_periods,
         std::unique_ptr<aggregation> const& agg,
         agg_op device_agg_op,
         rmm::mr::device_memory_resource* mr,
         hipStream_t stream)
  {
    CUDF_FAIL(
      "Aggregation operator and/or input type combination is invalid: "
      "LEAD/LAG supported only on fixed-width types");
  }

  template <aggregation::Kind op,
            typename PrecedingWindowIterator,
            typename FollowingWindowIterator>
  std::enable_if_t<!(op == aggregation::MEAN || op == aggregation::LEAD || op == aggregation::LAG),
                   std::unique_ptr<column>>
  operator()(column_view const& input,
             column_view const& default_outputs,
             PrecedingWindowIterator preceding_window_begin,
             FollowingWindowIterator following_window_begin,
             size_type min_periods,
             std::unique_ptr<aggregation> const& agg,
             rmm::mr::device_memory_resource* mr,
             hipStream_t stream)
  {
    CUDF_EXPECTS(default_outputs.is_empty(),
                 "Only LEAD/LAG window functions support default values.");

    return launch<InputType,
                  typename corresponding_operator<op>::type,
                  op,
                  PrecedingWindowIterator,
                  FollowingWindowIterator>(input,
                                           default_outputs,
                                           preceding_window_begin,
                                           following_window_begin,
                                           min_periods,
                                           agg,
                                           mr,
                                           stream);
  }

  // This variant is just to handle mean
  template <aggregation::Kind op,
            typename PrecedingWindowIterator,
            typename FollowingWindowIterator>
  std::enable_if_t<(op == aggregation::MEAN), std::unique_ptr<column>> operator()(
    column_view const& input,
    column_view const& default_outputs,
    PrecedingWindowIterator preceding_window_begin,
    FollowingWindowIterator following_window_begin,
    size_type min_periods,
    std::unique_ptr<aggregation> const& agg,
    rmm::mr::device_memory_resource* mr,
    hipStream_t stream)
  {
    return launch<InputType, cudf::DeviceSum, op, PrecedingWindowIterator, FollowingWindowIterator>(
      input,
      default_outputs,
      preceding_window_begin,
      following_window_begin,
      min_periods,
      agg,
      mr,
      stream);
  }

  template <aggregation::Kind op,
            typename PrecedingWindowIterator,
            typename FollowingWindowIterator>
  std::enable_if_t<(op == aggregation::LEAD || op == aggregation::LAG), std::unique_ptr<column>>
  operator()(column_view const& input,
             column_view const& default_outputs,
             PrecedingWindowIterator preceding_window_begin,
             FollowingWindowIterator following_window_begin,
             size_type min_periods,
             std::unique_ptr<aggregation> const& agg,
             rmm::mr::device_memory_resource* mr,
             hipStream_t stream)
  {
    return launch<InputType,
                  cudf::DeviceLeadLag,
                  op,
                  PrecedingWindowIterator,
                  FollowingWindowIterator>(
      input,
      default_outputs,
      preceding_window_begin,
      following_window_begin,
      min_periods,
      agg,
      cudf::DeviceLeadLag{static_cast<cudf::detail::lead_lag_aggregation*>(agg.get())->row_offset},
      mr,
      stream);
  }
};

struct dispatch_rolling {
  template <typename T, typename PrecedingWindowIterator, typename FollowingWindowIterator>
  std::unique_ptr<column> operator()(column_view const& input,
                                     column_view const& default_outputs,
                                     PrecedingWindowIterator preceding_window_begin,
                                     FollowingWindowIterator following_window_begin,
                                     size_type min_periods,
                                     std::unique_ptr<aggregation> const& agg,
                                     rmm::mr::device_memory_resource* mr,
                                     hipStream_t stream)
  {
    return aggregation_dispatcher(agg->kind,
                                  rolling_window_launcher<T>{},
                                  input,
                                  default_outputs,
                                  preceding_window_begin,
                                  following_window_begin,
                                  min_periods,
                                  agg,
                                  mr,
                                  stream);
  }
};

}  // namespace

// Applies a user-defined rolling window function to the values in a column.
template <typename PrecedingWindowIterator, typename FollowingWindowIterator>
std::unique_ptr<column> rolling_window_udf(column_view const& input,

                                           PrecedingWindowIterator preceding_window,
                                           std::string const& preceding_window_str,
                                           FollowingWindowIterator following_window,
                                           std::string const& following_window_str,
                                           size_type min_periods,
                                           std::unique_ptr<aggregation> const& agg,
                                           rmm::mr::device_memory_resource* mr,
                                           hipStream_t stream = 0)
{
  static_assert(warp_size == cudf::detail::size_in_bits<cudf::bitmask_type>(),
                "bitmask_type size does not match CUDA warp size");

  if (input.has_nulls())
    CUDF_FAIL("Currently the UDF version of rolling window does NOT support inputs with nulls.");

  min_periods = std::max(min_periods, 0);

  auto udf_agg = static_cast<udf_aggregation*>(agg.get());

  std::string hash = "prog_rolling." + std::to_string(std::hash<std::string>{}(udf_agg->_source));

  std::string cuda_source;
  switch (udf_agg->kind) {
    case aggregation::Kind::PTX:
      cuda_source = cudf::rolling::jit::code::kernel_headers;
      cuda_source +=
        cudf::jit::parse_single_function_ptx(udf_agg->_source,
                                             udf_agg->_function_name,
                                             cudf::jit::get_type_name(udf_agg->_output_type),
                                             {0, 5});  // args 0 and 5 are pointers.
      cuda_source += cudf::rolling::jit::code::kernel;
      break;
    case aggregation::Kind::CUDA:
      cuda_source = cudf::rolling::jit::code::kernel_headers;
      cuda_source +=
        cudf::jit::parse_single_function_cuda(udf_agg->_source, udf_agg->_function_name);
      cuda_source += cudf::rolling::jit::code::kernel;
      break;
    default: CUDF_FAIL("Unsupported UDF type.");
  }

  std::unique_ptr<column> output = make_numeric_column(
    udf_agg->_output_type, input.size(), cudf::mask_state::UNINITIALIZED, stream, mr);

  auto output_view = output->mutable_view();
  rmm::device_scalar<size_type> device_valid_count{0, stream};

  const std::vector<std::string> compiler_flags{"-std=c++14",
                                                // Have jitify prune unused global variables
                                                "-remove-unused-globals",
                                                // suppress all NVRTC warnings
                                                "-w"};

  // Launch the jitify kernel
  cudf::jit::launcher(hash,
                      cuda_source,
                      {cudf_types_hpp,
                       cudf_utilities_bit_hpp,
                       cudf::rolling::jit::code::operation_h,
                       ___src_rolling_rolling_jit_detail_hpp},
                      compiler_flags,
                      nullptr,
                      stream)
    .set_kernel_inst("gpu_rolling_new",  // name of the kernel we are launching
                     {cudf::jit::get_type_name(input.type()),  // list of template arguments
                      cudf::jit::get_type_name(output->type()),
                      udf_agg->_operator_name,
                      preceding_window_str.c_str(),
                      following_window_str.c_str()})
    .launch(input.size(),
            cudf::jit::get_data_ptr(input),
            input.null_mask(),
            cudf::jit::get_data_ptr(output_view),
            output_view.null_mask(),
            device_valid_count.data(),
            preceding_window,
            following_window,
            min_periods);

  output->set_null_count(output->size() - device_valid_count.value(stream));

  // check the stream for debugging
  CHECK_CUDA(stream);

  return output;
}

/**
 * @copydoc cudf::rolling_window(column_view const& input,
 *                               PrecedingWindowIterator preceding_window_begin,
 *                               FollowingWindowIterator following_window_begin,
 *                               size_type min_periods,
 *                               std::unique_ptr<aggregation> const& agg,
 *                               rmm::mr::device_memory_resource* mr)
 *
 * @param stream CUDA stream used for device memory operations and kernel launches.
 */
template <typename PrecedingWindowIterator, typename FollowingWindowIterator>
std::unique_ptr<column> rolling_window(column_view const& input,
                                       column_view const& default_outputs,
                                       PrecedingWindowIterator preceding_window_begin,
                                       FollowingWindowIterator following_window_begin,
                                       size_type min_periods,
                                       std::unique_ptr<aggregation> const& agg,
                                       rmm::mr::device_memory_resource* mr,
                                       hipStream_t stream = 0)
{
  static_assert(warp_size == cudf::detail::size_in_bits<cudf::bitmask_type>(),
                "bitmask_type size does not match CUDA warp size");

  min_periods = std::max(min_periods, 0);

  return cudf::type_dispatcher(input.type(),
                               dispatch_rolling{},
                               input,
                               default_outputs,
                               preceding_window_begin,
                               following_window_begin,
                               min_periods,
                               agg,
                               mr,
                               stream);
}

}  // namespace detail

// Applies a fixed-size rolling window function to the values in a column.
std::unique_ptr<column> rolling_window(column_view const& input,
                                       size_type preceding_window,
                                       size_type following_window,
                                       size_type min_periods,
                                       std::unique_ptr<aggregation> const& agg,
                                       rmm::mr::device_memory_resource* mr)
{
  return rolling_window(
    input, empty_like(input)->view(), preceding_window, following_window, min_periods, agg, mr);
}

// Applies a fixed-size rolling window function to the values in a column.
std::unique_ptr<column> rolling_window(column_view const& input,
                                       column_view const& default_outputs,
                                       size_type preceding_window,
                                       size_type following_window,
                                       size_type min_periods,
                                       std::unique_ptr<aggregation> const& agg,
                                       rmm::mr::device_memory_resource* mr)
{
  CUDF_FUNC_RANGE();

  if (input.is_empty()) return empty_like(input);
  CUDF_EXPECTS((min_periods >= 0), "min_periods must be non-negative");

  CUDF_EXPECTS((default_outputs.is_empty() || default_outputs.size() == input.size()),
               "Defaults column must be either empty or have as many rows as the input column.");

  if (agg->kind == aggregation::CUDA || agg->kind == aggregation::PTX) {
    return cudf::detail::rolling_window_udf(input,
                                            preceding_window,
                                            "cudf::size_type",
                                            following_window,
                                            "cudf::size_type",
                                            min_periods,
                                            agg,
                                            mr,
                                            0);
  } else {
    auto preceding_window_begin = thrust::make_constant_iterator(preceding_window);
    auto following_window_begin = thrust::make_constant_iterator(following_window);

    return cudf::detail::rolling_window(input,
                                        default_outputs,
                                        preceding_window_begin,
                                        following_window_begin,
                                        min_periods,
                                        agg,
                                        mr,
                                        0);
  }
}

// Applies a variable-size rolling window function to the values in a column.
std::unique_ptr<column> rolling_window(column_view const& input,
                                       column_view const& preceding_window,
                                       column_view const& following_window,
                                       size_type min_periods,
                                       std::unique_ptr<aggregation> const& agg,
                                       rmm::mr::device_memory_resource* mr)
{
  CUDF_FUNC_RANGE();

  if (preceding_window.is_empty() || following_window.is_empty() || input.is_empty())
    return empty_like(input);

  CUDF_EXPECTS(preceding_window.type().id() == type_id::INT32 &&
                 following_window.type().id() == type_id::INT32,
               "preceding_window/following_window must have type_id::INT32 type");

  CUDF_EXPECTS(preceding_window.size() == input.size() && following_window.size() == input.size(),
               "preceding_window/following_window size must match input size");

  if (agg->kind == aggregation::CUDA || agg->kind == aggregation::PTX) {
    return cudf::detail::rolling_window_udf(input,
                                            preceding_window.begin<size_type>(),
                                            "cudf::size_type*",
                                            following_window.begin<size_type>(),
                                            "cudf::size_type*",
                                            min_periods,
                                            agg,
                                            mr,
                                            0);
  } else {
    return cudf::detail::rolling_window(input,
                                        empty_like(input)->view(),
                                        preceding_window.begin<size_type>(),
                                        following_window.begin<size_type>(),
                                        min_periods,
                                        agg,
                                        mr,
                                        0);
  }
}

std::unique_ptr<column> grouped_rolling_window(table_view const& group_keys,
                                               column_view const& input,
                                               size_type preceding_window,
                                               size_type following_window,
                                               size_type min_periods,
                                               std::unique_ptr<aggregation> const& aggr,
                                               rmm::mr::device_memory_resource* mr)
{
  return grouped_rolling_window(group_keys,
                                input,
                                empty_like(input)->view(),
                                preceding_window,
                                following_window,
                                min_periods,
                                aggr,
                                mr);
}

std::unique_ptr<column> grouped_rolling_window(table_view const& group_keys,
                                               column_view const& input,
                                               column_view const& default_outputs,
                                               size_type preceding_window,
                                               size_type following_window,
                                               size_type min_periods,
                                               std::unique_ptr<aggregation> const& aggr,
                                               rmm::mr::device_memory_resource* mr)
{
  CUDF_FUNC_RANGE();

  if (input.is_empty()) return empty_like(input);

  CUDF_EXPECTS((group_keys.num_columns() == 0 || group_keys.num_rows() == input.size()),
               "Size mismatch between group_keys and input vector.");

  CUDF_EXPECTS((min_periods > 0), "min_periods must be positive");

  CUDF_EXPECTS((default_outputs.is_empty() || default_outputs.size() == input.size()),
               "Defaults column must be either empty or have as many rows as the input column.");

  if (group_keys.num_columns() == 0) {
    // No Groupby columns specified. Treat as one big group.
    return rolling_window(
      input, default_outputs, preceding_window, following_window, min_periods, aggr, mr);
  }

  using sort_groupby_helper = cudf::groupby::detail::sort::sort_groupby_helper;

  sort_groupby_helper helper{group_keys, cudf::null_policy::INCLUDE, cudf::sorted::YES};
  auto group_offsets{helper.group_offsets()};
  auto const& group_labels{helper.group_labels()};

  // `group_offsets` are interpreted in adjacent pairs, each pair representing the offsets
  // of the first, and one past the last elements in a group.
  //
  // If `group_offsets` is not empty, it must contain at least two offsets:
  //   a. 0, indicating the first element in `input`
  //   b. input.size(), indicating one past the last element in `input`.
  //
  // Thus, for an input of 1000 rows,
  //   0. [] indicates a single group, spanning the entire column.
  //   1  [10] is invalid.
  //   2. [0, 1000] indicates a single group, spanning the entire column (thus, equivalent to no
  //   groups.)
  //   3. [0, 500, 1000] indicates two equal-sized groups: [0,500), and [500,1000).

  assert(group_offsets.size() >= 2 && group_offsets[0] == 0 &&
         group_offsets[group_offsets.size() - 1] == input.size() &&
         "Must have at least one group.");

  auto preceding_calculator = [d_group_offsets = group_offsets.data().get(),
                               d_group_labels  = group_labels.data().get(),
                               preceding_window] __device__(size_type idx) {
    auto group_label = d_group_labels[idx];
    auto group_start = d_group_offsets[group_label];
    return thrust::minimum<size_type>{}(preceding_window,
                                        idx - group_start + 1);  // Preceding includes current row.
  };

  auto following_calculator = [d_group_offsets = group_offsets.data().get(),
                               d_group_labels  = group_labels.data().get(),
                               following_window] __device__(size_type idx) {
    auto group_label = d_group_labels[idx];
    auto group_end =
      d_group_offsets[group_label +
                      1];  // Cannot fall off the end, since offsets is capped with `input.size()`.
    return thrust::minimum<size_type>{}(following_window, (group_end - 1) - idx);
  };

  if (aggr->kind == aggregation::CUDA || aggr->kind == aggregation::PTX) {
    cudf::detail::preceding_window_wrapper grouped_preceding_window{
      group_offsets.data().get(), group_labels.data().get(), preceding_window};

    cudf::detail::following_window_wrapper grouped_following_window{
      group_offsets.data().get(), group_labels.data().get(), following_window};

    return cudf::detail::rolling_window_udf(input,
                                            grouped_preceding_window,
                                            "cudf::detail::preceding_window_wrapper",
                                            grouped_following_window,
                                            "cudf::detail::following_window_wrapper",
                                            min_periods,
                                            aggr,
                                            mr,
                                            0);
  } else {
    return cudf::detail::rolling_window(
      input,
      default_outputs,
      thrust::make_transform_iterator(thrust::make_counting_iterator<size_type>(0),
                                      preceding_calculator),
      thrust::make_transform_iterator(thrust::make_counting_iterator<size_type>(0),
                                      following_calculator),
      min_periods,
      aggr,
      mr,
      0);
  }
}

namespace {

bool is_supported_range_frame_unit(cudf::data_type const& data_type)
{
  auto id = data_type.id();
  return id == cudf::type_id::TIMESTAMP_DAYS || id == cudf::type_id::TIMESTAMP_SECONDS ||
         id == cudf::type_id::TIMESTAMP_MILLISECONDS ||
         id == cudf::type_id::TIMESTAMP_MICROSECONDS || id == cudf::type_id::TIMESTAMP_NANOSECONDS;
}

/// Fetches multiplication factor to normalize window sizes, depending on the datatype of the
/// timestamp column. Used for time-based rolling-window operations. E.g. If the timestamp column is
/// in TIMESTAMP_SECONDS, and the window sizes are specified in DAYS, the window size needs to be
/// multiplied by `24*60*60`, before comparisons with the timestamps.
size_t multiplication_factor(cudf::data_type const& data_type)
{
  // Assume timestamps.
  switch (data_type.id()) {
    case cudf::type_id::TIMESTAMP_DAYS: return 1L;
    case cudf::type_id::TIMESTAMP_SECONDS: return 24L * 60 * 60;
    case cudf::type_id::TIMESTAMP_MILLISECONDS: return 24L * 60 * 60 * 1000;
    case cudf::type_id::TIMESTAMP_MICROSECONDS: return 24L * 60 * 60 * 1000 * 1000;
    default:
      CUDF_EXPECTS(data_type.id() == cudf::type_id::TIMESTAMP_NANOSECONDS,
                   "Unexpected data-type for timestamp-based rolling window operation!");
      return 24L * 60 * 60 * 1000 * 1000 * 1000;
  }
}

/// Given a single, ungrouped timestamp column, return the indices corresponding
/// to the first null timestamp, and (one past) the last null timestamp.
/// The input column is sorted, with all null values clustered either
/// at the beginning of the column or at the end.
/// If no null values are founds, null_begin and null_end are 0.
std::tuple<size_type, size_type> get_null_bounds_for_timestamp_column(
  column_view const& timestamp_column)
{
  auto const num_rows  = timestamp_column.size();
  auto const num_nulls = timestamp_column.null_count();

  if (num_nulls == num_rows || num_nulls == 0) {
    // Short-circuit: All nulls, or no nulls.
    return std::make_tuple(0, num_nulls);
  }

  auto const first_row_is_null = timestamp_column.null_count(0, 1) == 1;

  return first_row_is_null ? std::make_tuple(0, num_nulls)
                           : std::make_tuple(num_rows - num_nulls, num_rows);
}

/// Time-range window computation, with
///   1. no grouping keys specified
///   2. timetamps in ASCENDING order.
/// Treat as one single group.
template <typename TimeT>
std::unique_ptr<column> time_range_window_ASC(column_view const& input,
                                              column_view const& timestamp_column,
                                              TimeT preceding_window,
                                              TimeT following_window,
                                              size_type min_periods,
                                              std::unique_ptr<aggregation> const& aggr,
                                              rmm::mr::device_memory_resource* mr)
{
  size_type nulls_begin_idx, nulls_end_idx;
  std::tie(nulls_begin_idx, nulls_end_idx) = get_null_bounds_for_timestamp_column(timestamp_column);

  auto preceding_calculator = [nulls_begin_idx,
                               nulls_end_idx,
                               d_timestamps = timestamp_column.data<TimeT>(),
                               preceding_window] __device__(size_type idx) -> size_type {
    if (idx >= nulls_begin_idx && idx < nulls_end_idx) {
      // Current row is in the null group.
      // Must consider beginning of null-group as window start.
      return idx - nulls_begin_idx + 1;
    }

    // timestamp[idx] not null. Binary search the group, excluding null group.
    // If nulls_begin_idx == 0, either
    //  1. NULLS FIRST ordering: Binary search starts where nulls_end_idx.
    //  2. NO NULLS: Binary search starts at 0 (also nulls_end_idx).
    // Otherwise, NULLS LAST ordering. Start at 0.
    auto group_start                = nulls_begin_idx == 0 ? nulls_end_idx : 0;
    auto lowest_timestamp_in_window = d_timestamps[idx] - preceding_window;

    return ((d_timestamps + idx) - thrust::lower_bound(thrust::seq,
                                                       d_timestamps + group_start,
                                                       d_timestamps + idx,
                                                       lowest_timestamp_in_window)) +
           1;  // Add 1, for `preceding` to account for current row.
  };

  auto following_calculator = [nulls_begin_idx,
                               nulls_end_idx,
                               num_rows     = input.size(),
                               d_timestamps = timestamp_column.data<TimeT>(),
                               following_window] __device__(size_type idx) -> size_type {
    if (idx >= nulls_begin_idx && idx < nulls_end_idx) {
      // Current row is in the null group.
      // Window ends at the end of the null group.
      return nulls_end_idx - idx - 1;
    }

    // timestamp[idx] not null. Binary search the group, excluding null group.
    // If nulls_begin_idx == 0, either
    //  1. NULLS FIRST ordering: Binary search ends at num_rows.
    //  2. NO NULLS: Binary search also ends at num_rows.
    // Otherwise, NULLS LAST ordering. End at nulls_begin_idx.

    auto group_end                   = nulls_begin_idx == 0 ? num_rows : nulls_begin_idx;
    auto highest_timestamp_in_window = d_timestamps[idx] + following_window;

    return (thrust::upper_bound(thrust::seq,
                                d_timestamps + idx,
                                d_timestamps + group_end,
                                highest_timestamp_in_window) -
            (d_timestamps + idx)) -
           1;
  };

  return cudf::detail::rolling_window(
    input,
    empty_like(input)->view(),
    thrust::make_transform_iterator(thrust::make_counting_iterator<size_type>(0),
                                    preceding_calculator),
    thrust::make_transform_iterator(thrust::make_counting_iterator<size_type>(0),
                                    following_calculator),
    min_periods,
    aggr,
    mr);
}

/// Given a timestamp column grouped as specified in group_offsets,
/// return the following two vectors:
///  1. Vector with one entry per group, indicating the offset in the group
///     where the null values begin.
///  2. Vector with one entry per group, indicating the offset in the group
///     where the null values end. (i.e. 1 past the last null.)
/// Each group in the input timestamp column must be sorted,
/// with null values clustered at either the start or the end of each group.
/// If there are no nulls for any given group, (nulls_begin, nulls_end) == (0,0).
std::tuple<rmm::device_vector<size_type>, rmm::device_vector<size_type>>
get_null_bounds_for_timestamp_column(column_view const& timestamp_column,
                                     rmm::device_vector<size_type> const& group_offsets)
{
  // For each group, the null values are themselves clustered
  // at the beginning or the end of the group.
  // These nulls cannot participate, except in their own window.

  // If the input has n groups, group_offsets will have n+1 values.
  // null_start and null_end should eventually have 1 entry per group.
  auto null_start = rmm::device_vector<size_type>(group_offsets.begin(), group_offsets.end() - 1);
  auto null_end   = rmm::device_vector<size_type>(group_offsets.begin(), group_offsets.end() - 1);

  if (timestamp_column.has_nulls()) {
    auto p_timestamps_device_view = column_device_view::create(timestamp_column);
    auto num_groups               = group_offsets.size();

    // Null timestamps exist. Find null bounds, per group.
    thrust::for_each(
      thrust::device,
      thrust::make_counting_iterator(static_cast<size_type>(0)),
      thrust::make_counting_iterator(static_cast<size_type>(num_groups)),
      [d_timestamps    = *p_timestamps_device_view,
       d_group_offsets = group_offsets.data().get(),
       d_null_start    = null_start.data(),
       d_null_end      = null_end.data()] __device__(auto group_label) {
        auto group_start           = d_group_offsets[group_label];
        auto group_end             = d_group_offsets[group_label + 1];
        auto first_element_is_null = d_timestamps.is_null_nocheck(group_start);
        auto last_element_is_null  = d_timestamps.is_null_nocheck(group_end - 1);
        if (!first_element_is_null && !last_element_is_null) {
          // Short circuit: No nulls.
          d_null_start[group_label] = group_start;
          d_null_end[group_label]   = group_start;
        } else if (first_element_is_null && last_element_is_null) {
          // Short circuit: All nulls.
          d_null_start[group_label] = group_start;
          d_null_end[group_label]   = group_end;
        } else if (first_element_is_null) {
          // NULLS FIRST.
          d_null_start[group_label] = group_start;
          d_null_end[group_label]   = *thrust::partition_point(
            thrust::seq,
            thrust::make_counting_iterator(group_start),
            thrust::make_counting_iterator(group_end),
            [&d_timestamps] __device__(auto i) { return d_timestamps.is_null_nocheck(i); });
        } else {
          // NULLS LAST.
          d_null_end[group_label]   = group_end;
          d_null_start[group_label] = *thrust::partition_point(
            thrust::seq,
            thrust::make_counting_iterator(group_start),
            thrust::make_counting_iterator(group_end),
            [&d_timestamps] __device__(auto i) { return d_timestamps.is_valid_nocheck(i); });
        }
      });
  }

  return std::make_tuple(std::move(null_start), std::move(null_end));
}

// Time-range window computation, for timestamps in ASCENDING order.
template <typename TimeT>
std::unique_ptr<column> time_range_window_ASC(
  column_view const& input,
  column_view const& timestamp_column,
  rmm::device_vector<cudf::size_type> const& group_offsets,
  rmm::device_vector<cudf::size_type> const& group_labels,
  TimeT preceding_window,
  TimeT following_window,
  size_type min_periods,
  std::unique_ptr<aggregation> const& aggr,
  rmm::mr::device_memory_resource* mr)
{
  rmm::device_vector<size_type> null_start, null_end;
  std::tie(null_start, null_end) =
    get_null_bounds_for_timestamp_column(timestamp_column, group_offsets);

  auto preceding_calculator = [d_group_offsets = group_offsets.data().get(),
                               d_group_labels  = group_labels.data().get(),
                               d_timestamps    = timestamp_column.data<TimeT>(),
                               d_nulls_begin   = null_start.data().get(),
                               d_nulls_end     = null_end.data().get(),
                               preceding_window] __device__(size_type idx) -> size_type {
    auto group_label = d_group_labels[idx];
    auto group_start = d_group_offsets[group_label];
    auto nulls_begin = d_nulls_begin[group_label];
    auto nulls_end   = d_nulls_end[group_label];

    // If idx lies in the null-range, the window is the null range.
    if (idx >= nulls_begin && idx < nulls_end) {
      // Current row is in the null group.
      // The window starts at the start of the null group.
      return idx - nulls_begin + 1;
    }

    // timestamp[idx] not null. Search must exclude the null group.
    // If nulls_begin == group_start, either of the following is true:
    //  1. NULLS FIRST ordering: Search must begin at nulls_end.
    //  2. NO NULLS: Search must begin at group_start (which also equals nulls_end.)
    // Otherwise, NULLS LAST ordering. Search must start at nulls group_start.
    auto search_start = nulls_begin == group_start ? nulls_end : group_start;

    auto lowest_timestamp_in_window = d_timestamps[idx] - preceding_window;

    return ((d_timestamps + idx) - thrust::lower_bound(thrust::seq,
                                                       d_timestamps + search_start,
                                                       d_timestamps + idx,
                                                       lowest_timestamp_in_window)) +
           1;  // Add 1, for `preceding` to account for current row.
  };

  auto following_calculator = [d_group_offsets = group_offsets.data().get(),
                               d_group_labels  = group_labels.data().get(),
                               d_timestamps    = timestamp_column.data<TimeT>(),
                               d_nulls_begin   = null_start.data().get(),
                               d_nulls_end     = null_end.data().get(),
                               following_window] __device__(size_type idx) -> size_type {
    auto group_label = d_group_labels[idx];
    auto group_start = d_group_offsets[group_label];
    auto group_end =
      d_group_offsets[group_label +
                      1];  // Cannot fall off the end, since offsets is capped with `input.size()`.
    auto nulls_begin = d_nulls_begin[group_label];
    auto nulls_end   = d_nulls_end[group_label];

    // If idx lies in the null-range, the window is the null range.
    if (idx >= nulls_begin && idx < nulls_end) {
      // Current row is in the null group.
      // The window ends at the end of the null group.
      return nulls_end - idx - 1;
    }

    // timestamp[idx] not null. Search must exclude the null group.
    // If nulls_begin == group_start, either of the following is true:
    //  1. NULLS FIRST ordering: Search ends at group_end.
    //  2. NO NULLS: Search ends at group_end.
    // Otherwise, NULLS LAST ordering. Search ends at nulls_begin.
    auto search_end = nulls_begin == group_start ? group_end : nulls_begin;

    auto highest_timestamp_in_window = d_timestamps[idx] + following_window;

    return (thrust::upper_bound(thrust::seq,
                                d_timestamps + idx,
                                d_timestamps + search_end,
                                highest_timestamp_in_window) -
            (d_timestamps + idx)) -
           1;
  };

  return cudf::detail::rolling_window(
    input,
    empty_like(input)->view(),
    thrust::make_transform_iterator(thrust::make_counting_iterator<size_type>(0),
                                    preceding_calculator),
    thrust::make_transform_iterator(thrust::make_counting_iterator<size_type>(0),
                                    following_calculator),
    min_periods,
    aggr,
    mr);
}

/// Time-range window computation, with
///   1. no grouping keys specified
///   2. timetamps in DESCENDING order.
/// Treat as one single group.
template <typename TimeT>
std::unique_ptr<column> time_range_window_DESC(column_view const& input,
                                               column_view const& timestamp_column,
                                               TimeT preceding_window,
                                               TimeT following_window,
                                               size_type min_periods,
                                               std::unique_ptr<aggregation> const& aggr,
                                               rmm::mr::device_memory_resource* mr)
{
  size_type nulls_begin_idx, nulls_end_idx;
  std::tie(nulls_begin_idx, nulls_end_idx) = get_null_bounds_for_timestamp_column(timestamp_column);

  auto preceding_calculator = [nulls_begin_idx,
                               nulls_end_idx,
                               d_timestamps = timestamp_column.data<TimeT>(),
                               preceding_window] __device__(size_type idx) -> size_type {
    if (idx >= nulls_begin_idx && idx < nulls_end_idx) {
      // Current row is in the null group.
      // Must consider beginning of null-group as window start.
      return idx - nulls_begin_idx + 1;
    }

    // timestamp[idx] not null. Binary search the group, excluding null group.
    // If nulls_begin_idx == 0, either
    //  1. NULLS FIRST ordering: Binary search starts where nulls_end_idx.
    //  2. NO NULLS: Binary search starts at 0 (also nulls_end_idx).
    // Otherwise, NULLS LAST ordering. Start at 0.
    auto group_start                 = nulls_begin_idx == 0 ? nulls_end_idx : 0;
    auto highest_timestamp_in_window = d_timestamps[idx] + preceding_window;

    return ((d_timestamps + idx) -
            thrust::lower_bound(thrust::seq,
                                d_timestamps + group_start,
                                d_timestamps + idx,
                                highest_timestamp_in_window,
                                thrust::greater<decltype(highest_timestamp_in_window)>())) +
           1;  // Add 1, for `preceding` to account for current row.
  };

  auto following_calculator = [nulls_begin_idx,
                               nulls_end_idx,
                               num_rows     = input.size(),
                               d_timestamps = timestamp_column.data<TimeT>(),
                               following_window] __device__(size_type idx) -> size_type {
    if (idx >= nulls_begin_idx && idx < nulls_end_idx) {
      // Current row is in the null group.
      // Window ends at the end of the null group.
      return nulls_end_idx - idx - 1;
    }

    // timestamp[idx] not null. Search must exclude null group.
    // If nulls_begin_idx = 0, either
    //  1. NULLS FIRST ordering: Search ends at num_rows.
    //  2. NO NULLS: Search also ends at num_rows.
    // Otherwise, NULLS LAST ordering: End at nulls_begin_idx.

    auto group_end                  = nulls_begin_idx == 0 ? num_rows : nulls_begin_idx;
    auto lowest_timestamp_in_window = d_timestamps[idx] - following_window;

    return (thrust::upper_bound(thrust::seq,
                                d_timestamps + idx,
                                d_timestamps + group_end,
                                lowest_timestamp_in_window,
                                thrust::greater<decltype(lowest_timestamp_in_window)>()) -
            (d_timestamps + idx)) -
           1;
  };

  return cudf::detail::rolling_window(
    input,
    empty_like(input)->view(),
    thrust::make_transform_iterator(thrust::make_counting_iterator<size_type>(0),
                                    preceding_calculator),
    thrust::make_transform_iterator(thrust::make_counting_iterator<size_type>(0),
                                    following_calculator),
    min_periods,
    aggr,
    mr);
}

// Time-range window computation, for timestamps in DESCENDING order.
template <typename TimestampImpl_t>
std::unique_ptr<column> time_range_window_DESC(
  column_view const& input,
  column_view const& timestamp_column,
  rmm::device_vector<cudf::size_type> const& group_offsets,
  rmm::device_vector<cudf::size_type> const& group_labels,
  TimestampImpl_t preceding_window,
  TimestampImpl_t following_window,
  size_type min_periods,
  std::unique_ptr<aggregation> const& aggr,
  rmm::mr::device_memory_resource* mr)
{
  rmm::device_vector<size_type> null_start, null_end;
  std::tie(null_start, null_end) =
    get_null_bounds_for_timestamp_column(timestamp_column, group_offsets);

  auto preceding_calculator = [d_group_offsets = group_offsets.data().get(),
                               d_group_labels  = group_labels.data().get(),
                               d_timestamps    = timestamp_column.data<TimestampImpl_t>(),
                               d_nulls_begin   = null_start.data().get(),
                               d_nulls_end     = null_end.data().get(),
                               preceding_window] __device__(size_type idx) -> size_type {
    auto group_label = d_group_labels[idx];
    auto group_start = d_group_offsets[group_label];
    auto nulls_begin = d_nulls_begin[group_label];
    auto nulls_end   = d_nulls_end[group_label];

    // If idx lies in the null-range, the window is the null range.
    if (idx >= nulls_begin && idx < nulls_end) {
      // Current row is in the null group.
      // The window starts at the start of the null group.
      return idx - nulls_begin + 1;
    }

    // timestamp[idx] not null. Search must exclude the null group.
    // If nulls_begin == group_start, either of the following is true:
    //  1. NULLS FIRST ordering: Search must begin at nulls_end.
    //  2. NO NULLS: Search must begin at group_start (which also equals nulls_end.)
    // Otherwise, NULLS LAST ordering. Search must start at nulls group_start.
    auto search_start = nulls_begin == group_start ? nulls_end : group_start;

    auto highest_timestamp_in_window = d_timestamps[idx] + preceding_window;

    return ((d_timestamps + idx) -
            thrust::lower_bound(thrust::seq,
                                d_timestamps + search_start,
                                d_timestamps + idx,
                                highest_timestamp_in_window,
                                thrust::greater<decltype(highest_timestamp_in_window)>())) +
           1;  // Add 1, for `preceding` to account for current row.
  };

  auto following_calculator = [d_group_offsets = group_offsets.data().get(),
                               d_group_labels  = group_labels.data().get(),
                               d_timestamps    = timestamp_column.data<TimestampImpl_t>(),
                               d_nulls_begin   = null_start.data().get(),
                               d_nulls_end     = null_end.data().get(),
                               following_window] __device__(size_type idx) -> size_type {
    auto group_label = d_group_labels[idx];
    auto group_start = d_group_offsets[group_label];
    auto group_end   = d_group_offsets[group_label + 1];
    auto nulls_begin = d_nulls_begin[group_label];
    auto nulls_end   = d_nulls_end[group_label];

    // If idx lies in the null-range, the window is the null range.
    if (idx >= nulls_begin && idx < nulls_end) {
      // Current row is in the null group.
      // The window ends at the end of the null group.
      return nulls_end - idx - 1;
    }

    // timestamp[idx] not null. Search must exclude the null group.
    // If nulls_begin == group_start, either of the following is true:
    //  1. NULLS FIRST ordering: Search ends at group_end.
    //  2. NO NULLS: Search ends at group_end.
    // Otherwise, NULLS LAST ordering. Search ends at nulls_begin.
    auto search_end = nulls_begin == group_start ? group_end : nulls_begin;

    auto lowest_timestamp_in_window = d_timestamps[idx] - following_window;

    return (thrust::upper_bound(thrust::seq,
                                d_timestamps + idx,
                                d_timestamps + search_end,
                                lowest_timestamp_in_window,
                                thrust::greater<decltype(lowest_timestamp_in_window)>()) -
            (d_timestamps + idx)) -
           1;
  };

  if (aggr->kind == aggregation::CUDA || aggr->kind == aggregation::PTX) {
    CUDF_FAIL("Time ranged rolling window does NOT (yet) support UDF.");
  } else {
    return cudf::detail::rolling_window(
      input,
      empty_like(input)->view(),
      thrust::make_transform_iterator(thrust::make_counting_iterator<size_type>(0),
                                      preceding_calculator),
      thrust::make_transform_iterator(thrust::make_counting_iterator<size_type>(0),
                                      following_calculator),
      min_periods,
      aggr,
      mr,
      0);
  }
}

template <typename TimestampImpl_t>
std::unique_ptr<column> grouped_time_range_rolling_window_impl(
  column_view const& input,
  column_view const& timestamp_column,
  cudf::order const& timestamp_ordering,
  rmm::device_vector<cudf::size_type> const& group_offsets,
  rmm::device_vector<cudf::size_type> const& group_labels,
  size_type preceding_window_in_days,  // TODO: Consider taking offset-type as type_id. Assumes days
                                       // for now.
  size_type following_window_in_days,
  size_type min_periods,
  std::unique_ptr<aggregation> const& aggr,
  rmm::mr::device_memory_resource* mr)
{
  TimestampImpl_t mult_factor{
    static_cast<TimestampImpl_t>(multiplication_factor(timestamp_column.type()))};

  if (timestamp_ordering == cudf::order::ASCENDING) {
    return group_offsets.empty() ? time_range_window_ASC(input,
                                                         timestamp_column,
                                                         preceding_window_in_days * mult_factor,
                                                         following_window_in_days * mult_factor,
                                                         min_periods,
                                                         aggr,
                                                         mr)
                                 : time_range_window_ASC(input,
                                                         timestamp_column,
                                                         group_offsets,
                                                         group_labels,
                                                         preceding_window_in_days * mult_factor,
                                                         following_window_in_days * mult_factor,
                                                         min_periods,
                                                         aggr,
                                                         mr);
  } else {
    return group_offsets.empty() ? time_range_window_DESC(input,
                                                          timestamp_column,
                                                          preceding_window_in_days * mult_factor,
                                                          following_window_in_days * mult_factor,
                                                          min_periods,
                                                          aggr,
                                                          mr)
                                 : time_range_window_DESC(input,
                                                          timestamp_column,
                                                          group_offsets,
                                                          group_labels,
                                                          preceding_window_in_days * mult_factor,
                                                          following_window_in_days * mult_factor,
                                                          min_periods,
                                                          aggr,
                                                          mr);
  }
}

}  // namespace

std::unique_ptr<column> grouped_time_range_rolling_window(table_view const& group_keys,
                                                          column_view const& timestamp_column,
                                                          cudf::order const& timestamp_order,
                                                          column_view const& input,
                                                          size_type preceding_window_in_days,
                                                          size_type following_window_in_days,
                                                          size_type min_periods,
                                                          std::unique_ptr<aggregation> const& aggr,
                                                          rmm::mr::device_memory_resource* mr)
{
  CUDF_FUNC_RANGE();

  if (input.is_empty()) return empty_like(input);

  CUDF_EXPECTS((group_keys.num_columns() == 0 || group_keys.num_rows() == input.size()),
               "Size mismatch between group_keys and input vector.");

  CUDF_EXPECTS((min_periods > 0), "min_periods must be positive");

  using sort_groupby_helper = cudf::groupby::detail::sort::sort_groupby_helper;
  using index_vector        = sort_groupby_helper::index_vector;

  index_vector group_offsets, group_labels;
  if (group_keys.num_columns() > 0) {
    sort_groupby_helper helper{group_keys, cudf::null_policy::INCLUDE, cudf::sorted::YES};
    group_offsets = helper.group_offsets();
    group_labels  = helper.group_labels();
  }

  // Assumes that `timestamp_column` is actually of a timestamp type.
  CUDF_EXPECTS(is_supported_range_frame_unit(timestamp_column.type()),
               "Unsupported data-type for `timestamp`-based rolling window operation!");

  return timestamp_column.type().id() == cudf::type_id::TIMESTAMP_DAYS
           ? grouped_time_range_rolling_window_impl<int32_t>(input,
                                                             timestamp_column,
                                                             timestamp_order,
                                                             group_offsets,
                                                             group_labels,
                                                             preceding_window_in_days,
                                                             following_window_in_days,
                                                             min_periods,
                                                             aggr,
                                                             mr)
           : grouped_time_range_rolling_window_impl<int64_t>(input,
                                                             timestamp_column,
                                                             timestamp_order,
                                                             group_offsets,
                                                             group_labels,
                                                             preceding_window_in_days,
                                                             following_window_in_days,
                                                             min_periods,
                                                             aggr,
                                                             mr);
}

}  // namespace cudf
