#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019-2020, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <cudf/aggregation.hpp>
#include <cudf/column/column_device_view.cuh>
#include <cudf/column/column_factories.hpp>
#include <cudf/column/column_view.hpp>
#include <cudf/copying.hpp>
#include <cudf/detail/aggregation/aggregation.cuh>
#include <cudf/detail/copy.hpp>
#include <cudf/detail/gather.hpp>
#include <cudf/detail/groupby/sort_helper.hpp>
#include <cudf/detail/nvtx/ranges.hpp>
#include <cudf/detail/utilities/cuda.cuh>
#include <cudf/rolling.hpp>
#include <cudf/types.hpp>
#include <cudf/utilities/bit.hpp>
#include <rolling/rolling_detail.hpp>
#include <rolling/rolling_jit_detail.hpp>

#include <jit/launcher.h>
#include <jit/parser.h>
#include <jit/type.h>
#include <rolling/jit/code/code.h>

#include <bit.hpp.jit>
#include <rolling_jit_detail.hpp.jit>
#include <types.hpp.jit>

#include <thrust/binary_search.h>
#include <rmm/device_scalar.hpp>

#include <memory>

namespace cudf {
namespace detail {
namespace {  // anonymous
/**
 * @brief Only count operation is executed and count is updated
 *        depending on `min_periods` and returns true if it was
 *        valid, else false.
 */
template <typename InputType,
          typename OutputType,
          typename agg_op,
          aggregation::Kind op,
          bool has_nulls>
std::enable_if_t<op == aggregation::COUNT_VALID || op == aggregation::COUNT_ALL, bool> __device__
process_rolling_window(column_device_view input,
                       mutable_column_device_view output,
                       size_type start_index,
                       size_type end_index,
                       size_type current_index,
                       size_type min_periods)
{
  // declare this as volatile to avoid some compiler optimizations that lead to incorrect results
  // for CUDA 10.0 and below (fixed in CUDA 10.1)
  volatile cudf::size_type count = 0;

  for (size_type j = start_index; j < end_index; j++) {
    if (op == aggregation::COUNT_ALL || !has_nulls || input.is_valid(j)) { count++; }
  }

  bool output_is_valid                      = (count >= min_periods);
  output.element<OutputType>(current_index) = count;

  return output_is_valid;
}

/**
 * @brief Calculates row-number within [start_index, end_index).
 *        Count is updated depending on `min_periods`
 *        Returns true if it was valid, else false.
 */
template <typename InputType,
          typename OutputType,
          typename agg_op,
          aggregation::Kind op,
          bool has_nulls>
std::enable_if_t<op == aggregation::ROW_NUMBER, bool> __device__
process_rolling_window(column_device_view input,
                       mutable_column_device_view output,
                       size_type start_index,
                       size_type end_index,
                       size_type current_index,
                       size_type min_periods)
{
  bool output_is_valid                      = ((end_index - start_index) >= min_periods);
  output.element<OutputType>(current_index) = ((current_index - start_index) + 1);

  return output_is_valid;
}

/**
 * @brief Only used for `string_view` type to get ARGMIN and ARGMAX, which
 *        will be used to gather MIN and MAX. And returns true if the
 *        operation was valid, else false.
 */
template <typename InputType,
          typename OutputType,
          typename agg_op,
          aggregation::Kind op,
          bool has_nulls>
std::enable_if_t<(op == aggregation::ARGMIN or op == aggregation::ARGMAX) and
                   std::is_same<InputType, cudf::string_view>::value,
                 bool>
  __device__ process_rolling_window(column_device_view input,
                                    mutable_column_device_view output,
                                    size_type start_index,
                                    size_type end_index,
                                    size_type current_index,
                                    size_type min_periods)
{
  // declare this as volatile to avoid some compiler optimizations that lead to incorrect results
  // for CUDA 10.0 and below (fixed in CUDA 10.1)
  volatile cudf::size_type count = 0;
  InputType val                  = agg_op::template identity<InputType>();
  OutputType val_index           = (op == aggregation::ARGMIN) ? ARGMIN_SENTINEL : ARGMAX_SENTINEL;

  for (size_type j = start_index; j < end_index; j++) {
    if (!has_nulls || input.is_valid(j)) {
      InputType element = input.element<InputType>(j);
      val               = agg_op{}(element, val);
      if (val == element) { val_index = j; }
      count++;
    }
  }

  bool output_is_valid = (count >= min_periods);
  // -1 will help identify null elements while gathering for Min and Max
  // In case of count, this would be null, so doesn't matter.
  output.element<OutputType>(current_index) = (output_is_valid) ? val_index : -1;

  // The gather mask shouldn't contain null values, so
  // always return zero
  return true;
}

/**
 * @brief Operates on only fixed-width types and returns true if the
 *        operation was valid, else false.
 */
template <typename InputType,
          typename OutputType,
          typename agg_op,
          aggregation::Kind op,
          bool has_nulls>
std::enable_if_t<!std::is_same<InputType, cudf::string_view>::value and
                   !(op == aggregation::COUNT_VALID || op == aggregation::COUNT_ALL ||
                     op == aggregation::ROW_NUMBER),
                 bool>
  __device__ process_rolling_window(column_device_view input,
                                    mutable_column_device_view output,
                                    size_type start_index,
                                    size_type end_index,
                                    size_type current_index,
                                    size_type min_periods)
{
  // declare this as volatile to avoid some compiler optimizations that lead to incorrect results
  // for CUDA 10.0 and below (fixed in CUDA 10.1)
  volatile cudf::size_type count = 0;
  OutputType val                 = agg_op::template identity<OutputType>();

  for (size_type j = start_index; j < end_index; j++) {
    if (!has_nulls || input.is_valid(j)) {
      OutputType element = input.element<InputType>(j);
      val                = agg_op{}(element, val);
      count++;
    }
  }

  bool output_is_valid = (count >= min_periods);

  // store the output value, one per thread
  cudf::detail::rolling_store_output_functor<OutputType, op == aggregation::MEAN>{}(
    output.element<OutputType>(current_index), val, count);

  return output_is_valid;
}

/**
 * @brief Computes the rolling window function
 *
 * @tparam InputType  Datatype of `input`
 * @tparam OutputType  Datatype of `output`
 * @tparam agg_op  A functor that defines the aggregation operation
 * @tparam op The aggregation operator (enum value)
 * @tparam block_size CUDA block size for the kernel
 * @tparam has_nulls true if the input column has nulls
 * @tparam PrecedingWindowIterator iterator type (inferred)
 * @tparam FollowingWindowIterator iterator type (inferred)
 * @param input Input column device view
 * @param output Output column device view
 * @param preceding_window_begin[in] Rolling window size iterator, accumulates from
 *                in_col[i-preceding_window] to in_col[i] inclusive
 * @param following_window_begin[in] Rolling window size iterator in the forward
 *                direction, accumulates from in_col[i] to
 *                in_col[i+following_window] inclusive
 * @param min_periods[in]  Minimum number of observations in window required to
 *                have a value, otherwise 0 is stored in the valid bit mask
 */
template <typename InputType,
          typename OutputType,
          typename agg_op,
          aggregation::Kind op,
          int block_size,
          bool has_nulls,
          typename PrecedingWindowIterator,
          typename FollowingWindowIterator>
__launch_bounds__(block_size) __global__
  void gpu_rolling(column_device_view input,
                   mutable_column_device_view output,
                   size_type* __restrict__ output_valid_count,
                   PrecedingWindowIterator preceding_window_begin,
                   FollowingWindowIterator following_window_begin,
                   size_type min_periods)
{
  size_type i      = blockIdx.x * block_size + threadIdx.x;
  size_type stride = block_size * gridDim.x;

  size_type warp_valid_count{0};

  auto active_threads = __ballot_sync(0xffffffff, i < input.size());
  while (i < input.size()) {
    size_type preceding_window = preceding_window_begin[i];
    size_type following_window = following_window_begin[i];

    // compute bounds
    size_type start       = min(input.size(), max(0, i - preceding_window + 1));
    size_type end         = min(input.size(), max(0, i + following_window + 1));
    size_type start_index = min(start, end);
    size_type end_index   = max(start, end);

    // aggregate
    // TODO: We should explore using shared memory to avoid redundant loads.
    //       This might require separating the kernel into a special version
    //       for dynamic and static sizes.

    volatile bool output_is_valid = false;
    output_is_valid = process_rolling_window<InputType, OutputType, agg_op, op, has_nulls>(
      input, output, start_index, end_index, i, min_periods);

    // set the mask
    cudf::bitmask_type result_mask{__ballot_sync(active_threads, output_is_valid)};

    // only one thread writes the mask
    if (0 == threadIdx.x % cudf::detail::warp_size) {
      output.set_mask_word(cudf::word_index(i), result_mask);
      warp_valid_count += __popc(result_mask);
    }

    // process next element
    i += stride;
    active_threads = __ballot_sync(active_threads, i < input.size());
  }

  // sum the valid counts across the whole block
  size_type block_valid_count =
    cudf::detail::single_lane_block_sum_reduce<block_size, 0>(warp_valid_count);

  if (threadIdx.x == 0) { atomicAdd(output_valid_count, block_valid_count); }
}

template <typename InputType>
struct rolling_window_launcher {
  template <typename T,
            typename agg_op,
            aggregation::Kind op,
            typename PrecedingWindowIterator,
            typename FollowingWindowIterator>
  size_type kernel_launcher(column_view const& input,
                            mutable_column_view& output,
                            PrecedingWindowIterator preceding_window_begin,
                            FollowingWindowIterator following_window_begin,
                            size_type min_periods,
                            std::unique_ptr<aggregation> const& agg,
                            hipStream_t stream)
  {
    constexpr cudf::size_type block_size = 256;
    cudf::detail::grid_1d grid(input.size(), block_size);

    auto input_device_view  = column_device_view::create(input, stream);
    auto output_device_view = mutable_column_device_view::create(output, stream);

    rmm::device_scalar<size_type> device_valid_count{0, stream};

    if (input.has_nulls()) {
      gpu_rolling<T, target_type_t<InputType, op>, agg_op, op, block_size, true>
        <<<grid.num_blocks, block_size, 0, stream>>>(*input_device_view,
                                                     *output_device_view,
                                                     device_valid_count.data(),
                                                     preceding_window_begin,
                                                     following_window_begin,
                                                     min_periods);
    } else {
      gpu_rolling<T, target_type_t<InputType, op>, agg_op, op, block_size, false>
        <<<grid.num_blocks, block_size, 0, stream>>>(*input_device_view,
                                                     *output_device_view,
                                                     device_valid_count.data(),
                                                     preceding_window_begin,
                                                     following_window_begin,
                                                     min_periods);
    }

    size_type valid_count = device_valid_count.value(stream);

    // check the stream for debugging
    CHECK_CUDA(stream);

    return valid_count;
  }

  // This launch is only for fixed width columns with valid aggregation option
  // numeric: All
  // timestamp: MIN, MAX, COUNT_VALID, COUNT_ALL, ROW_NUMBER
  // string, dictionary, list : COUNT_VALID, COUNT_ALL, ROW_NUMBER
  template <typename T,
            typename agg_op,
            aggregation::Kind op,
            typename PrecedingWindowIterator,
            typename FollowingWindowIterator>
  std::enable_if_t<cudf::detail::is_rolling_supported<T, agg_op, op>() and
                     !cudf::detail::is_rolling_string_specialization<T, agg_op, op>(),
                   std::unique_ptr<column>>
  launch(column_view const& input,
         PrecedingWindowIterator preceding_window_begin,
         FollowingWindowIterator following_window_begin,
         size_type min_periods,
         std::unique_ptr<aggregation> const& agg,
         rmm::mr::device_memory_resource* mr,
         hipStream_t stream)
  {
    if (input.is_empty()) return empty_like(input);

    auto output = make_fixed_width_column(
      target_type(input.type(), op), input.size(), mask_state::UNINITIALIZED, stream, mr);

    cudf::mutable_column_view output_view = output->mutable_view();
    auto valid_count =
      kernel_launcher<T, agg_op, op, PrecedingWindowIterator, FollowingWindowIterator>(
        input,
        output_view,
        preceding_window_begin,
        following_window_begin,
        min_periods,
        agg,
        stream);

    output->set_null_count(output->size() - valid_count);

    return output;
  }

  // This launch is only for string specializations
  // string: MIN, MAX
  template <typename T,
            typename agg_op,
            aggregation::Kind op,
            typename PrecedingWindowIterator,
            typename FollowingWindowIterator>
  std::enable_if_t<cudf::detail::is_rolling_string_specialization<T, agg_op, op>(),
                   std::unique_ptr<column>>
  launch(column_view const& input,
         PrecedingWindowIterator preceding_window_begin,
         FollowingWindowIterator following_window_begin,
         size_type min_periods,
         std::unique_ptr<aggregation> const& agg,
         rmm::mr::device_memory_resource* mr,
         hipStream_t stream)
  {
    if (input.is_empty()) return empty_like(input);

    auto output = make_numeric_column(cudf::data_type{cudf::type_to_id<size_type>()},
                                      input.size(),
                                      cudf::mask_state::UNINITIALIZED,
                                      stream,
                                      mr);

    cudf::mutable_column_view output_view = output->mutable_view();

    // Passing the agg_op and aggregation::Kind as constant to group them in pair, else it
    // evolves to error when try to use agg_op as compiler tries different combinations
    if (op == aggregation::MIN) {
      kernel_launcher<T,
                      DeviceMin,
                      aggregation::ARGMIN,
                      PrecedingWindowIterator,
                      FollowingWindowIterator>(input,
                                               output_view,
                                               preceding_window_begin,
                                               following_window_begin,
                                               min_periods,
                                               agg,
                                               stream);
    } else if (op == aggregation::MAX) {
      kernel_launcher<T,
                      DeviceMax,
                      aggregation::ARGMAX,
                      PrecedingWindowIterator,
                      FollowingWindowIterator>(input,
                                               output_view,
                                               preceding_window_begin,
                                               following_window_begin,
                                               min_periods,
                                               agg,
                                               stream);
    } else {
      CUDF_FAIL("MIN and MAX are the only supported aggregation types for string columns");
    }

    // The rows that represent null elements will be having negative values in gather map,
    // and that's why nullify_out_of_bounds/ignore_out_of_bounds is true.
    auto output_table = detail::gather(table_view{{input}},
                                       output->view(),
                                       detail::out_of_bounds_policy::IGNORE,
                                       detail::negative_index_policy::NOT_ALLOWED,
                                       mr,
                                       stream);
    return std::make_unique<cudf::column>(std::move(output_table->get_column(0)));
  }

  // Deals with invalid column and/or aggregation options
  template <typename T,
            typename agg_op,
            aggregation::Kind op,
            typename PrecedingWindowIterator,
            typename FollowingWindowIterator>
  std::enable_if_t<!cudf::detail::is_rolling_supported<T, agg_op, op>() and
                     !cudf::detail::is_rolling_string_specialization<T, agg_op, op>(),
                   std::unique_ptr<column>>
  launch(column_view const& input,
         PrecedingWindowIterator preceding_window_begin,
         FollowingWindowIterator following_window_begin,
         size_type min_periods,
         std::unique_ptr<aggregation> const& agg,
         rmm::mr::device_memory_resource* mr,
         hipStream_t stream)
  {
    CUDF_FAIL("Aggregation operator and/or input type combination is invalid");
  }

  template <aggregation::Kind op,
            typename PrecedingWindowIterator,
            typename FollowingWindowIterator>
  std::enable_if_t<!(op == aggregation::MEAN), std::unique_ptr<column>> operator()(
    column_view const& input,
    PrecedingWindowIterator preceding_window_begin,
    FollowingWindowIterator following_window_begin,
    size_type min_periods,
    std::unique_ptr<aggregation> const& agg,
    rmm::mr::device_memory_resource* mr,
    hipStream_t stream)
  {
    return launch<InputType,
                  typename corresponding_operator<op>::type,
                  op,
                  PrecedingWindowIterator,
                  FollowingWindowIterator>(
      input, preceding_window_begin, following_window_begin, min_periods, agg, mr, stream);
  }

  // This variant is just to handle mean
  template <aggregation::Kind op,
            typename PrecedingWindowIterator,
            typename FollowingWindowIterator>
  std::enable_if_t<(op == aggregation::MEAN), std::unique_ptr<column>> operator()(
    column_view const& input,
    PrecedingWindowIterator preceding_window_begin,
    FollowingWindowIterator following_window_begin,
    size_type min_periods,
    std::unique_ptr<aggregation> const& agg,
    rmm::mr::device_memory_resource* mr,
    hipStream_t stream)
  {
    return launch<InputType, cudf::DeviceSum, op, PrecedingWindowIterator, FollowingWindowIterator>(
      input, preceding_window_begin, following_window_begin, min_periods, agg, mr, stream);
  }
};

struct dispatch_rolling {
  template <typename T, typename PrecedingWindowIterator, typename FollowingWindowIterator>
  std::unique_ptr<column> operator()(column_view const& input,
                                     PrecedingWindowIterator preceding_window_begin,
                                     FollowingWindowIterator following_window_begin,
                                     size_type min_periods,
                                     std::unique_ptr<aggregation> const& agg,
                                     rmm::mr::device_memory_resource* mr,
                                     hipStream_t stream)
  {
    return aggregation_dispatcher(agg->kind,
                                  rolling_window_launcher<T>{},
                                  input,
                                  preceding_window_begin,
                                  following_window_begin,
                                  min_periods,
                                  agg,
                                  mr,
                                  stream);
  }
};

}  // namespace

// Applies a user-defined rolling window function to the values in a column.
template <typename PrecedingWindowIterator, typename FollowingWindowIterator>
std::unique_ptr<column> rolling_window_udf(column_view const& input,

                                           PrecedingWindowIterator preceding_window,
                                           std::string const& preceding_window_str,
                                           FollowingWindowIterator following_window,
                                           std::string const& following_window_str,
                                           size_type min_periods,
                                           std::unique_ptr<aggregation> const& agg,
                                           rmm::mr::device_memory_resource* mr,
                                           hipStream_t stream = 0)
{
  static_assert(warp_size == cudf::detail::size_in_bits<cudf::bitmask_type>(),
                "bitmask_type size does not match CUDA warp size");

  if (input.has_nulls())
    CUDF_FAIL("Currently the UDF version of rolling window does NOT support inputs with nulls.");

  min_periods = std::max(min_periods, 0);

  auto udf_agg = static_cast<udf_aggregation*>(agg.get());

  std::string hash = "prog_rolling." + std::to_string(std::hash<std::string>{}(udf_agg->_source));

  std::string cuda_source;
  switch (udf_agg->kind) {
    case aggregation::Kind::PTX:
      cuda_source = cudf::rolling::jit::code::kernel_headers;
      cuda_source +=
        cudf::jit::parse_single_function_ptx(udf_agg->_source,
                                             udf_agg->_function_name,
                                             cudf::jit::get_type_name(udf_agg->_output_type),
                                             {0, 5});  // args 0 and 5 are pointers.
      cuda_source += cudf::rolling::jit::code::kernel;
      break;
    case aggregation::Kind::CUDA:
      cuda_source = cudf::rolling::jit::code::kernel_headers;
      cuda_source +=
        cudf::jit::parse_single_function_cuda(udf_agg->_source, udf_agg->_function_name);
      cuda_source += cudf::rolling::jit::code::kernel;
      break;
    default: CUDF_FAIL("Unsupported UDF type.");
  }

  std::unique_ptr<column> output = make_numeric_column(
    udf_agg->_output_type, input.size(), cudf::mask_state::UNINITIALIZED, stream, mr);

  auto output_view = output->mutable_view();
  rmm::device_scalar<size_type> device_valid_count{0, stream};

  const std::vector<std::string> compiler_flags{"-std=c++14",
                                                // Have jitify prune unused global variables
                                                "-remove-unused-globals",
                                                // suppress all NVRTC warnings
                                                "-w"};

  // Launch the jitify kernel
  cudf::jit::launcher(hash,
                      cuda_source,
                      {cudf_types_hpp,
                       cudf_utilities_bit_hpp,
                       cudf::rolling::jit::code::operation_h,
                       ___src_rolling_rolling_jit_detail_hpp},
                      compiler_flags,
                      nullptr,
                      stream)
    .set_kernel_inst("gpu_rolling_new",  // name of the kernel we are launching
                     {cudf::jit::get_type_name(input.type()),  // list of template arguments
                      cudf::jit::get_type_name(output->type()),
                      udf_agg->_operator_name,
                      preceding_window_str.c_str(),
                      following_window_str.c_str()})
    .launch(input.size(),
            cudf::jit::get_data_ptr(input),
            input.null_mask(),
            cudf::jit::get_data_ptr(output_view),
            output_view.null_mask(),
            device_valid_count.data(),
            preceding_window,
            following_window,
            min_periods);

  output->set_null_count(output->size() - device_valid_count.value(stream));

  // check the stream for debugging
  CHECK_CUDA(stream);

  return output;
}

/**
 * @copydoc cudf::rolling_window(column_view const& input,
 *                               PrecedingWindowIterator preceding_window_begin,
 *                               FollowingWindowIterator following_window_begin,
 *                               size_type min_periods,
 *                               std::unique_ptr<aggregation> const& agg,
 *                               rmm::mr::device_memory_resource* mr)
 *
 * @param stream CUDA stream used for device memory operations and kernel launches.
 */
template <typename PrecedingWindowIterator, typename FollowingWindowIterator>
std::unique_ptr<column> rolling_window(column_view const& input,
                                       PrecedingWindowIterator preceding_window_begin,
                                       FollowingWindowIterator following_window_begin,
                                       size_type min_periods,
                                       std::unique_ptr<aggregation> const& agg,
                                       rmm::mr::device_memory_resource* mr,
                                       hipStream_t stream = 0)
{
  static_assert(warp_size == cudf::detail::size_in_bits<cudf::bitmask_type>(),
                "bitmask_type size does not match CUDA warp size");

  min_periods = std::max(min_periods, 0);

  return cudf::type_dispatcher(input.type(),
                               dispatch_rolling{},
                               input,
                               preceding_window_begin,
                               following_window_begin,
                               min_periods,
                               agg,
                               mr,
                               stream);
}

}  // namespace detail

// Applies a fixed-size rolling window function to the values in a column.
std::unique_ptr<column> rolling_window(column_view const& input,
                                       size_type preceding_window,
                                       size_type following_window,
                                       size_type min_periods,
                                       std::unique_ptr<aggregation> const& agg,
                                       rmm::mr::device_memory_resource* mr)
{
  CUDF_FUNC_RANGE();

  if (input.size() == 0) return empty_like(input);
  CUDF_EXPECTS((min_periods >= 0), "min_periods must be non-negative");

  if (agg->kind == aggregation::CUDA || agg->kind == aggregation::PTX) {
    return cudf::detail::rolling_window_udf(input,
                                            preceding_window,
                                            "cudf::size_type",
                                            following_window,
                                            "cudf::size_type",
                                            min_periods,
                                            agg,
                                            mr,
                                            0);
  } else {
    auto preceding_window_begin = thrust::make_constant_iterator(preceding_window);
    auto following_window_begin = thrust::make_constant_iterator(following_window);

    return cudf::detail::rolling_window(
      input, preceding_window_begin, following_window_begin, min_periods, agg, mr, 0);
  }
}

// Applies a variable-size rolling window function to the values in a column.
std::unique_ptr<column> rolling_window(column_view const& input,
                                       column_view const& preceding_window,
                                       column_view const& following_window,
                                       size_type min_periods,
                                       std::unique_ptr<aggregation> const& agg,
                                       rmm::mr::device_memory_resource* mr)
{
  CUDF_FUNC_RANGE();

  if (preceding_window.size() == 0 || following_window.size() == 0 || input.size() == 0)
    return empty_like(input);

  CUDF_EXPECTS(preceding_window.type().id() == type_id::INT32 &&
                 following_window.type().id() == type_id::INT32,
               "preceding_window/following_window must have type_id::INT32 type");

  CUDF_EXPECTS(preceding_window.size() == input.size() && following_window.size() == input.size(),
               "preceding_window/following_window size must match input size");

  if (agg->kind == aggregation::CUDA || agg->kind == aggregation::PTX) {
    return cudf::detail::rolling_window_udf(input,
                                            preceding_window.begin<size_type>(),
                                            "cudf::size_type*",
                                            following_window.begin<size_type>(),
                                            "cudf::size_type*",
                                            min_periods,
                                            agg,
                                            mr,
                                            0);
  } else {
    return cudf::detail::rolling_window(input,
                                        preceding_window.begin<size_type>(),
                                        following_window.begin<size_type>(),
                                        min_periods,
                                        agg,
                                        mr,
                                        0);
  }
}

std::unique_ptr<column> grouped_rolling_window(table_view const& group_keys,
                                               column_view const& input,
                                               size_type preceding_window,
                                               size_type following_window,
                                               size_type min_periods,
                                               std::unique_ptr<aggregation> const& aggr,
                                               rmm::mr::device_memory_resource* mr)
{
  CUDF_FUNC_RANGE();

  if (input.size() == 0) return empty_like(input);

  CUDF_EXPECTS((group_keys.num_columns() == 0 || group_keys.num_rows() == input.size()),
               "Size mismatch between group_keys and input vector.");

  CUDF_EXPECTS((min_periods > 0), "min_periods must be positive");

  if (group_keys.num_columns() == 0) {
    // No Groupby columns specified. Treat as one big group.
    return rolling_window(input, preceding_window, following_window, min_periods, aggr, mr);
  }

  using sort_groupby_helper = cudf::groupby::detail::sort::sort_groupby_helper;

  sort_groupby_helper helper{group_keys, cudf::null_policy::INCLUDE, cudf::sorted::YES};
  auto group_offsets{helper.group_offsets()};
  auto const& group_labels{helper.group_labels()};

  // `group_offsets` are interpreted in adjacent pairs, each pair representing the offsets
  // of the first, and one past the last elements in a group.
  //
  // If `group_offsets` is not empty, it must contain at least two offsets:
  //   a. 0, indicating the first element in `input`
  //   b. input.size(), indicating one past the last element in `input`.
  //
  // Thus, for an input of 1000 rows,
  //   0. [] indicates a single group, spanning the entire column.
  //   1  [10] is invalid.
  //   2. [0, 1000] indicates a single group, spanning the entire column (thus, equivalent to no
  //   groups.)
  //   3. [0, 500, 1000] indicates two equal-sized groups: [0,500), and [500,1000).

  assert(group_offsets.size() >= 2 && group_offsets[0] == 0 &&
         group_offsets[group_offsets.size() - 1] == input.size() &&
         "Must have at least one group.");

  auto preceding_calculator = [d_group_offsets = group_offsets.data().get(),
                               d_group_labels  = group_labels.data().get(),
                               preceding_window] __device__(size_type idx) {
    auto group_label = d_group_labels[idx];
    auto group_start = d_group_offsets[group_label];
    return thrust::minimum<size_type>{}(preceding_window,
                                        idx - group_start + 1);  // Preceding includes current row.
  };

  auto following_calculator = [d_group_offsets = group_offsets.data().get(),
                               d_group_labels  = group_labels.data().get(),
                               following_window] __device__(size_type idx) {
    auto group_label = d_group_labels[idx];
    auto group_end =
      d_group_offsets[group_label +
                      1];  // Cannot fall off the end, since offsets is capped with `input.size()`.
    return thrust::minimum<size_type>{}(following_window, (group_end - 1) - idx);
  };

  if (aggr->kind == aggregation::CUDA || aggr->kind == aggregation::PTX) {
    cudf::detail::preceding_window_wrapper grouped_preceding_window{
      group_offsets.data().get(), group_labels.data().get(), preceding_window};

    cudf::detail::following_window_wrapper grouped_following_window{
      group_offsets.data().get(), group_labels.data().get(), following_window};

    return cudf::detail::rolling_window_udf(input,
                                            grouped_preceding_window,
                                            "cudf::detail::preceding_window_wrapper",
                                            grouped_following_window,
                                            "cudf::detail::following_window_wrapper",
                                            min_periods,
                                            aggr,
                                            mr,
                                            0);
  } else {
    return cudf::detail::rolling_window(
      input,
      thrust::make_transform_iterator(thrust::make_counting_iterator<size_type>(0),
                                      preceding_calculator),
      thrust::make_transform_iterator(thrust::make_counting_iterator<size_type>(0),
                                      following_calculator),
      min_periods,
      aggr,
      mr,
      0);
  }
}

namespace {

bool is_supported_range_frame_unit(cudf::data_type const& data_type)
{
  auto id = data_type.id();
  return id == cudf::type_id::TIMESTAMP_DAYS || id == cudf::type_id::TIMESTAMP_SECONDS ||
         id == cudf::type_id::TIMESTAMP_MILLISECONDS ||
         id == cudf::type_id::TIMESTAMP_MICROSECONDS || id == cudf::type_id::TIMESTAMP_NANOSECONDS;
}

/// Fetches multiplication factor to normalize window sizes, depending on the datatype of the
/// timestamp column. Used for time-based rolling-window operations. E.g. If the timestamp column is
/// in TIMESTAMP_SECONDS, and the window sizes are specified in DAYS, the window size needs to be
/// multiplied by `24*60*60`, before comparisons with the timestamps.
size_t multiplication_factor(cudf::data_type const& data_type)
{
  // Assume timestamps.
  switch (data_type.id()) {
    case cudf::type_id::TIMESTAMP_DAYS: return 1L;
    case cudf::type_id::TIMESTAMP_SECONDS: return 24L * 60 * 60;
    case cudf::type_id::TIMESTAMP_MILLISECONDS: return 24L * 60 * 60 * 1000;
    case cudf::type_id::TIMESTAMP_MICROSECONDS: return 24L * 60 * 60 * 1000 * 1000;
    default:
      CUDF_EXPECTS(data_type.id() == cudf::type_id::TIMESTAMP_NANOSECONDS,
                   "Unexpected data-type for timestamp-based rolling window operation!");
      return 24L * 60 * 60 * 1000 * 1000 * 1000;
  }
}

// Time-range window computation, with
//   1. no grouping keys specified
//   2. timetamps in ASCENDING order.
// Treat as one single group.
template <typename TimestampImpl_t>
std::unique_ptr<column> time_range_window_ASC(column_view const& input,
                                              column_view const& timestamp_column,
                                              TimestampImpl_t preceding_window,
                                              TimestampImpl_t following_window,
                                              size_type min_periods,
                                              std::unique_ptr<aggregation> const& aggr,
                                              rmm::mr::device_memory_resource* mr)
{
  auto preceding_calculator = [d_timestamps = timestamp_column.data<TimestampImpl_t>(),
                               preceding_window] __device__(size_type idx) {
    auto group_start                = 0;
    auto lowest_timestamp_in_window = d_timestamps[idx] - preceding_window;

    return ((d_timestamps + idx) - thrust::lower_bound(thrust::seq,
                                                       d_timestamps + group_start,
                                                       d_timestamps + idx,
                                                       lowest_timestamp_in_window)) +
           1;  // Add 1, for `preceding` to account for current row.
  };

  auto following_calculator = [num_rows     = input.size(),
                               d_timestamps = timestamp_column.data<TimestampImpl_t>(),
                               following_window] __device__(size_type idx) {
    auto group_end                   = num_rows;
    auto highest_timestamp_in_window = d_timestamps[idx] + following_window;

    return (thrust::upper_bound(thrust::seq,
                                d_timestamps + idx,
                                d_timestamps + group_end,
                                highest_timestamp_in_window) -
            (d_timestamps + idx)) -
           1;
  };

  return cudf::detail::rolling_window(
    input,
    thrust::make_transform_iterator(thrust::make_counting_iterator<size_type>(0),
                                    preceding_calculator),
    thrust::make_transform_iterator(thrust::make_counting_iterator<size_type>(0),
                                    following_calculator),
    min_periods,
    aggr,
    mr);
}

// Time-range window computation, for timestamps in ASCENDING order.
template <typename TimestampImpl_t>
std::unique_ptr<column> time_range_window_ASC(
  column_view const& input,
  column_view const& timestamp_column,
  rmm::device_vector<cudf::size_type> const& group_offsets,
  rmm::device_vector<cudf::size_type> const& group_labels,
  TimestampImpl_t preceding_window,
  TimestampImpl_t following_window,
  size_type min_periods,
  std::unique_ptr<aggregation> const& aggr,
  rmm::mr::device_memory_resource* mr)
{
  auto preceding_calculator = [d_group_offsets = group_offsets.data().get(),
                               d_group_labels  = group_labels.data().get(),
                               d_timestamps    = timestamp_column.data<TimestampImpl_t>(),
                               preceding_window] __device__(size_type idx) {
    auto group_label                = d_group_labels[idx];
    auto group_start                = d_group_offsets[group_label];
    auto lowest_timestamp_in_window = d_timestamps[idx] - preceding_window;

    return ((d_timestamps + idx) - thrust::lower_bound(thrust::seq,
                                                       d_timestamps + group_start,
                                                       d_timestamps + idx,
                                                       lowest_timestamp_in_window)) +
           1;  // Add 1, for `preceding` to account for current row.
  };

  auto following_calculator = [d_group_offsets = group_offsets.data().get(),
                               d_group_labels  = group_labels.data().get(),
                               d_timestamps    = timestamp_column.data<TimestampImpl_t>(),
                               following_window] __device__(size_type idx) {
    auto group_label = d_group_labels[idx];
    auto group_end =
      d_group_offsets[group_label +
                      1];  // Cannot fall off the end, since offsets is capped with `input.size()`.
    auto highest_timestamp_in_window = d_timestamps[idx] + following_window;

    return (thrust::upper_bound(thrust::seq,
                                d_timestamps + idx,
                                d_timestamps + group_end,
                                highest_timestamp_in_window) -
            (d_timestamps + idx)) -
           1;
  };

  return cudf::detail::rolling_window(
    input,
    thrust::make_transform_iterator(thrust::make_counting_iterator<size_type>(0),
                                    preceding_calculator),
    thrust::make_transform_iterator(thrust::make_counting_iterator<size_type>(0),
                                    following_calculator),
    min_periods,
    aggr,
    mr);
}

// Time-range window computation, with
//   1. no grouping keys specified
//   2. timetamps in DESCENDING order.
// Treat as one single group.
template <typename TimestampImpl_t>
std::unique_ptr<column> time_range_window_DESC(column_view const& input,
                                               column_view const& timestamp_column,
                                               TimestampImpl_t preceding_window,
                                               TimestampImpl_t following_window,
                                               size_type min_periods,
                                               std::unique_ptr<aggregation> const& aggr,
                                               rmm::mr::device_memory_resource* mr)
{
  auto preceding_calculator = [d_timestamps = timestamp_column.data<TimestampImpl_t>(),
                               preceding_window] __device__(size_type idx) {
    auto group_start                 = 0;
    auto highest_timestamp_in_window = d_timestamps[idx] + preceding_window;

    return ((d_timestamps + idx) -
            thrust::lower_bound(thrust::seq,
                                d_timestamps + group_start,
                                d_timestamps + idx,
                                highest_timestamp_in_window,
                                thrust::greater<decltype(highest_timestamp_in_window)>())) +
           1;  // Add 1, for `preceding` to account for current row.
  };

  auto following_calculator = [num_rows     = input.size(),
                               d_timestamps = timestamp_column.data<TimestampImpl_t>(),
                               following_window] __device__(size_type idx) {
    auto group_end =
      num_rows;  // Cannot fall off the end, since offsets is capped with `input.size()`.
    auto lowest_timestamp_in_window = d_timestamps[idx] - following_window;

    return (thrust::upper_bound(thrust::seq,
                                d_timestamps + idx,
                                d_timestamps + group_end,
                                lowest_timestamp_in_window,
                                thrust::greater<decltype(lowest_timestamp_in_window)>()) -
            (d_timestamps + idx)) -
           1;
  };

  return cudf::detail::rolling_window(
    input,
    thrust::make_transform_iterator(thrust::make_counting_iterator<size_type>(0),
                                    preceding_calculator),
    thrust::make_transform_iterator(thrust::make_counting_iterator<size_type>(0),
                                    following_calculator),
    min_periods,
    aggr,
    mr);
}

// Time-range window computation, for timestamps in DESCENDING order.
template <typename TimestampImpl_t>
std::unique_ptr<column> time_range_window_DESC(
  column_view const& input,
  column_view const& timestamp_column,
  rmm::device_vector<cudf::size_type> const& group_offsets,
  rmm::device_vector<cudf::size_type> const& group_labels,
  TimestampImpl_t preceding_window,
  TimestampImpl_t following_window,
  size_type min_periods,
  std::unique_ptr<aggregation> const& aggr,
  rmm::mr::device_memory_resource* mr)
{
  auto preceding_calculator = [d_group_offsets = group_offsets.data().get(),
                               d_group_labels  = group_labels.data().get(),
                               d_timestamps    = timestamp_column.data<TimestampImpl_t>(),
                               preceding_window] __device__(size_type idx) {
    auto group_label                 = d_group_labels[idx];
    auto group_start                 = d_group_offsets[group_label];
    auto highest_timestamp_in_window = d_timestamps[idx] + preceding_window;

    return ((d_timestamps + idx) -
            thrust::lower_bound(thrust::seq,
                                d_timestamps + group_start,
                                d_timestamps + idx,
                                highest_timestamp_in_window,
                                thrust::greater<decltype(highest_timestamp_in_window)>())) +
           1;  // Add 1, for `preceding` to account for current row.
  };

  auto following_calculator = [d_group_offsets = group_offsets.data().get(),
                               d_group_labels  = group_labels.data().get(),
                               d_timestamps    = timestamp_column.data<TimestampImpl_t>(),
                               following_window] __device__(size_type idx) {
    auto group_label = d_group_labels[idx];
    auto group_end =
      d_group_offsets[group_label +
                      1];  // Cannot fall off the end, since offsets is capped with `input.size()`.
    auto lowest_timestamp_in_window = d_timestamps[idx] - following_window;

    return (thrust::upper_bound(thrust::seq,
                                d_timestamps + idx,
                                d_timestamps + group_end,
                                lowest_timestamp_in_window,
                                thrust::greater<decltype(lowest_timestamp_in_window)>()) -
            (d_timestamps + idx)) -
           1;
  };

  if (aggr->kind == aggregation::CUDA || aggr->kind == aggregation::PTX) {
    CUDF_FAIL("Time ranged rolling window does NOT (yet) support UDF.");
  } else {
    return cudf::detail::rolling_window(
      input,
      thrust::make_transform_iterator(thrust::make_counting_iterator<size_type>(0),
                                      preceding_calculator),
      thrust::make_transform_iterator(thrust::make_counting_iterator<size_type>(0),
                                      following_calculator),
      min_periods,
      aggr,
      mr,
      0);
  }
}

template <typename TimestampImpl_t>
std::unique_ptr<column> grouped_time_range_rolling_window_impl(
  column_view const& input,
  column_view const& timestamp_column,
  cudf::order const& timestamp_ordering,
  rmm::device_vector<cudf::size_type> const& group_offsets,
  rmm::device_vector<cudf::size_type> const& group_labels,
  size_type preceding_window_in_days,  // TODO: Consider taking offset-type as type_id. Assumes days
                                       // for now.
  size_type following_window_in_days,
  size_type min_periods,
  std::unique_ptr<aggregation> const& aggr,
  rmm::mr::device_memory_resource* mr)
{
  TimestampImpl_t mult_factor{
    static_cast<TimestampImpl_t>(multiplication_factor(timestamp_column.type()))};

  if (timestamp_ordering == cudf::order::ASCENDING) {
    return (group_offsets.size() == 0)
             ? time_range_window_ASC(input,
                                     timestamp_column,
                                     preceding_window_in_days * mult_factor,
                                     following_window_in_days * mult_factor,
                                     min_periods,
                                     aggr,
                                     mr)
             : time_range_window_ASC(input,
                                     timestamp_column,
                                     group_offsets,
                                     group_labels,
                                     preceding_window_in_days * mult_factor,
                                     following_window_in_days * mult_factor,
                                     min_periods,
                                     aggr,
                                     mr);
  } else {
    return (group_offsets.size() == 0)
             ? time_range_window_DESC(input,
                                      timestamp_column,
                                      preceding_window_in_days * mult_factor,
                                      following_window_in_days * mult_factor,
                                      min_periods,
                                      aggr,
                                      mr)
             : time_range_window_DESC(input,
                                      timestamp_column,
                                      group_offsets,
                                      group_labels,
                                      preceding_window_in_days * mult_factor,
                                      following_window_in_days * mult_factor,
                                      min_periods,
                                      aggr,
                                      mr);
  }
}

}  // namespace

std::unique_ptr<column> grouped_time_range_rolling_window(table_view const& group_keys,
                                                          column_view const& timestamp_column,
                                                          cudf::order const& timestamp_order,
                                                          column_view const& input,
                                                          size_type preceding_window_in_days,
                                                          size_type following_window_in_days,
                                                          size_type min_periods,
                                                          std::unique_ptr<aggregation> const& aggr,
                                                          rmm::mr::device_memory_resource* mr)
{
  CUDF_FUNC_RANGE();

  if (input.size() == 0) return empty_like(input);

  CUDF_EXPECTS((group_keys.num_columns() == 0 || group_keys.num_rows() == input.size()),
               "Size mismatch between group_keys and input vector.");

  CUDF_EXPECTS((min_periods > 0), "min_periods must be positive");

  using sort_groupby_helper = cudf::groupby::detail::sort::sort_groupby_helper;
  using index_vector        = sort_groupby_helper::index_vector;

  index_vector group_offsets, group_labels;
  if (group_keys.num_columns() > 0) {
    sort_groupby_helper helper{group_keys, cudf::null_policy::INCLUDE, cudf::sorted::YES};
    group_offsets = helper.group_offsets();
    group_labels  = helper.group_labels();
  }

  // Assumes that `timestamp_column` is actually of a timestamp type.
  CUDF_EXPECTS(is_supported_range_frame_unit(timestamp_column.type()),
               "Unsupported data-type for `timestamp`-based rolling window operation!");

  return timestamp_column.type().id() == cudf::type_id::TIMESTAMP_DAYS
           ? grouped_time_range_rolling_window_impl<int32_t>(input,
                                                             timestamp_column,
                                                             timestamp_order,
                                                             group_offsets,
                                                             group_labels,
                                                             preceding_window_in_days,
                                                             following_window_in_days,
                                                             min_periods,
                                                             aggr,
                                                             mr)
           : grouped_time_range_rolling_window_impl<int64_t>(input,
                                                             timestamp_column,
                                                             timestamp_order,
                                                             group_offsets,
                                                             group_labels,
                                                             preceding_window_in_days,
                                                             following_window_in_days,
                                                             min_periods,
                                                             aggr,
                                                             mr);
}

}  // namespace cudf
