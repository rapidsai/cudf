#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019-2020, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <cudf/types.hpp>
#include <cudf/column/column_view.hpp>
#include <cudf/column/column_device_view.cuh>
#include <cudf/column/column_factories.hpp>
#include <cudf/detail/aggregation/aggregation.cuh>
#include <cudf/aggregation.hpp>
#include <rolling/rolling_detail.hpp>
#include <cudf/rolling.hpp>
#include <cudf/utilities/nvtx_utils.hpp>
#include <cudf/utilities/bit.hpp>
#include <cudf/detail/copy.hpp>
#include <cudf/detail/utilities/cuda.cuh>
#include <cudf/copying.hpp>

#include <jit/type.h>
#include <jit/launcher.h>
#include <jit/parser.h>
#include <rolling/jit/code/code.h>

#include <types.hpp.jit>
#include <bit.hpp.jit>

#include <rmm/device_scalar.hpp>

#include <memory>

namespace cudf {
namespace experimental {

namespace detail {

namespace { // anonymous

/**
 * @brief Computes the rolling window function
 *
 * @tparam ColumnType  Datatype of values pointed to by the pointers
 * @tparam agg_op  A functor that defines the aggregation operation
 * @tparam is_mean Compute mean=sum/count across all valid elements in the window
 * @tparam block_size CUDA block size for the kernel
 * @tparam has_nulls true if the input column has nulls
 * @tparam WindowIterator iterator type (inferred)
 * @param input Input column device view
 * @param output Output column device view
 * @param preceding_window_begin[in] Rolling window size iterator, accumulates from
 *                in_col[i-preceding_window] to in_col[i] inclusive
 * @param following_window_begin[in] Rolling window size iterator in the forward
 *                direction, accumulates from in_col[i] to
 *                in_col[i+following_window] inclusive
 * @param min_periods[in]  Minimum number of observations in window required to
 *                have a value, otherwise 0 is stored in the valid bit mask
 */
template <typename T, typename agg_op, aggregation::Kind op, int block_size, bool has_nulls,
          typename WindowIterator>
__launch_bounds__(block_size)
__global__
void gpu_rolling(column_device_view input,
                 mutable_column_device_view output,
                 size_type * __restrict__ output_valid_count,
                 WindowIterator preceding_window_begin,
                 WindowIterator following_window_begin,
                 size_type min_periods)
{
  size_type i = blockIdx.x * block_size + threadIdx.x;
  size_type stride = block_size * gridDim.x;

  size_type warp_valid_count{0};

  auto active_threads = __ballot_sync(0xffffffff, i < input.size());
  while(i < input.size())
  {
    T val = agg_op::template identity<T>();
    // declare this as volatile to avoid some compiler optimizations that lead to incorrect results
    // for CUDA 10.0 and below (fixed in CUDA 10.1)
    volatile cudf::size_type count = 0;

    size_type preceding_window = preceding_window_begin[i];
    size_type following_window = following_window_begin[i];

    // compute bounds
    size_type start = max(0, i - preceding_window);
    size_type end = min(input.size(), i + following_window + 1);
    size_type start_index = min(start, end);
    size_type end_index = max(start, end);

    // aggregate
    // TODO: We should explore using shared memory to avoid redundant loads.
    //       This might require separating the kernel into a special version
    //       for dynamic and static sizes.
    for (size_type j = start_index; j < end_index; j++) {
      if (!has_nulls || input.is_valid(j)) {
        // Element type and output type are different for COUNT
        T element = (op == aggregation::COUNT) ? T{0} : input.element<T>(j);
        val = agg_op{}(element, val);
        count++;
      }
    }

    // check if we have enough input samples
    bool output_is_valid = (count >= min_periods);

    // set the mask
    cudf::bitmask_type result_mask{__ballot_sync(active_threads, output_is_valid)};

    // only one thread writes the mask
    if (0 == threadIdx.x % cudf::experimental::detail::warp_size) {
      output.set_mask_word(cudf::word_index(i), result_mask);
      warp_valid_count += __popc(result_mask);
    }

    // store the output value, one per thread
    if (output_is_valid)
      cudf::detail::store_output_functor<T, op == aggregation::MEAN>{}(output.element<T>(i),
                                                                            val, count);
    // process next element 
    i += stride;
    active_threads = __ballot_sync(active_threads, i < input.size());
  }

  // sum the valid counts across the whole block  
  size_type block_valid_count = 
    cudf::experimental::detail::single_lane_block_sum_reduce<block_size, 0>(warp_valid_count);

  if(threadIdx.x == 0) {
    atomicAdd(output_valid_count, block_valid_count);
  }
}

template <typename InputType>
struct rolling_window_launcher
{

  template <typename T, typename agg_op, aggregation::Kind op, typename WindowIterator>
  std::enable_if_t<cudf::detail::is_supported<T, agg_op,
                                  op, op == aggregation::MEAN>(), std::unique_ptr<column>>
  launch(column_view const& input,
         WindowIterator preceding_window_begin,
         WindowIterator following_window_begin,
         size_type min_periods,
         std::unique_ptr<aggregation> const& agg,
         rmm::mr::device_memory_resource *mr,
         hipStream_t stream) {

    if (input.is_empty()) return empty_like(input);

    cudf::nvtx::range_push("CUDF_ROLLING_WINDOW", cudf::nvtx::color::ORANGE);

    min_periods = std::max(min_periods, 1);

    // output is always nullable, COUNT always INT32 output
    std::unique_ptr<column> output = (op == aggregation::COUNT) ?
        make_numeric_column(cudf::data_type{cudf::INT32}, input.size(),
                            cudf::UNINITIALIZED, stream, mr) :
        cudf::experimental::detail::allocate_like(input, input.size(),
          cudf::experimental::mask_allocation_policy::ALWAYS, mr, stream);

    constexpr cudf::size_type block_size = 256;
    cudf::experimental::detail::grid_1d grid(input.size(), block_size);

    auto input_device_view = column_device_view::create(input, stream);
    auto output_device_view = mutable_column_device_view::create(*output, stream);

    rmm::device_scalar<size_type> device_valid_count{0, stream};

    if (input.has_nulls()) {
        if (op == aggregation::COUNT) {
            gpu_rolling<size_type, agg_op, op, block_size, true><<<grid.num_blocks, block_size, 0, stream>>>
                (*input_device_view, *output_device_view, device_valid_count.data(),
                 preceding_window_begin, following_window_begin, min_periods);
        }
        else {
            gpu_rolling<InputType, agg_op, op, block_size, true><<<grid.num_blocks, block_size, 0, stream>>>
                (*input_device_view, *output_device_view, device_valid_count.data(),
                 preceding_window_begin, following_window_begin, min_periods);
        }
    } else {
        if (op == aggregation::COUNT) {
            gpu_rolling<size_type, agg_op, op, block_size, false><<<grid.num_blocks, block_size, 0, stream>>>
                (*input_device_view, *output_device_view, device_valid_count.data(),
                 preceding_window_begin, following_window_begin, min_periods);
        }
        else {
            gpu_rolling<InputType, agg_op, op, block_size, false><<<grid.num_blocks, block_size, 0, stream>>>
                (*input_device_view, *output_device_view, device_valid_count.data(),
                 preceding_window_begin, following_window_begin, min_periods);
        }
    }

    output->set_null_count(output->size() - device_valid_count.value(stream));

    // check the stream for debugging
    CHECK_CUDA(stream);

    cudf::nvtx::range_pop();

    return std::move(output);
  }

  template <typename T, typename agg_op, aggregation::Kind op, typename WindowIterator>
  std::enable_if_t<!cudf::detail::is_supported<T, agg_op,
                                  op, op == aggregation::MEAN>(), std::unique_ptr<column>>
  launch (column_view const& input,
          WindowIterator preceding_window_begin,
          WindowIterator following_window_begin,
          size_type min_periods,
          std::unique_ptr<aggregation> const& agg,
          rmm::mr::device_memory_resource *mr,
          hipStream_t stream) {
      CUDF_FAIL("Aggregation operator and/or input type combination is invalid");
  }


  template<aggregation::Kind op, typename WindowIterator>
  std::enable_if_t<!(op == aggregation::MEAN), std::unique_ptr<column>>
  operator()(column_view const& input,
                                     WindowIterator preceding_window_begin,
                                     WindowIterator following_window_begin,
                                     size_type min_periods,
                                     std::unique_ptr<aggregation> const& agg,
                                     rmm::mr::device_memory_resource *mr,
                                     hipStream_t stream)
  {
      return launch <InputType, typename corresponding_operator<op>::type, op, WindowIterator> (
              input,
              preceding_window_begin,
              following_window_begin,
              min_periods,
              agg,
              mr,
              stream);
  }

  template<aggregation::Kind op, typename WindowIterator>
  std::enable_if_t<(op == aggregation::MEAN), std::unique_ptr<column>>
  operator()(column_view const& input,
                                     WindowIterator preceding_window_begin,
                                     WindowIterator following_window_begin,
                                     size_type min_periods,
                                     std::unique_ptr<aggregation> const& agg,
                                     rmm::mr::device_memory_resource *mr,
                                     hipStream_t stream) {

      return launch <InputType, cudf::DeviceSum, op, WindowIterator> (
              input,
              preceding_window_begin,
              following_window_begin,
              min_periods,
              agg,
              mr,
              stream);
  }


};

struct dispatch_rolling {
    template <typename T, typename WindowIterator>
    std::unique_ptr<column> operator()(column_view const& input,
                                     WindowIterator preceding_window_begin,
                                     WindowIterator following_window_begin,
                                     size_type min_periods,
                                     std::unique_ptr<aggregation> const& agg,
                                     rmm::mr::device_memory_resource *mr,
                                     hipStream_t stream) {

        return aggregation_dispatcher(agg->kind, rolling_window_launcher<T>{},
                                      input,
                                      preceding_window_begin, following_window_begin,
                                      min_periods, agg, mr, stream);
    }
};

} // namespace anonymous

// Applies a user-defined rolling window function to the values in a column.
template <bool static_window, typename WindowIterator>
std::unique_ptr<column> rolling_window_udf(column_view const &input,
                                           WindowIterator preceding_window,
                                           WindowIterator following_window,
                                           size_type min_periods,
                                           std::unique_ptr<aggregation> const& agg,
                                           rmm::mr::device_memory_resource* mr,
                                           hipStream_t stream = 0)
{
  static_assert(warp_size == cudf::detail::size_in_bits<cudf::bitmask_type>(),
                "bitmask_type size does not match CUDA warp size");

  if (input.has_nulls())
    CUDF_FAIL("Currently the UDF version of rolling window does NOT support inputs with nulls.");

  cudf::nvtx::range_push("CUDF_ROLLING_WINDOW", cudf::nvtx::color::ORANGE);

  min_periods = std::max(min_periods, 1);

  auto udf_agg = static_cast<udf_aggregation*>(agg.get());

  std::string hash = "prog_experimental_rolling." 
    + std::to_string(std::hash<std::string>{}(udf_agg->_source));
  
  std::string cuda_source;
  switch(udf_agg->kind){
    case aggregation::Kind::PTX:
      cuda_source = cudf::experimental::rolling::jit::code::kernel_headers;
      cuda_source += cudf::jit::parse_single_function_ptx(udf_agg->_source, udf_agg->_function_name,
                                                          cudf::jit::get_type_name(udf_agg->_output_type),
                                                          {0, 5}); // args 0 and 5 are pointers.
      cuda_source += cudf::experimental::rolling::jit::code::kernel;
      break; 
    case aggregation::Kind::CUDA:
      cuda_source = cudf::experimental::rolling::jit::code::kernel_headers;
      cuda_source += cudf::jit::parse_single_function_cuda(udf_agg->_source, udf_agg->_function_name);
      cuda_source += cudf::experimental::rolling::jit::code::kernel;
      break;
    default:
      CUDF_FAIL("Unsupported UDF type.");
  }

  std::unique_ptr<column> output = make_numeric_column(udf_agg->_output_type, input.size(),
                                                       cudf::UNINITIALIZED, stream, mr);

  auto output_view = output->mutable_view();
  rmm::device_scalar<size_type> device_valid_count{0, stream};

  // Launch the jitify kernel
  cudf::jit::launcher(hash, cuda_source,
                      { cudf_types_hpp, cudf_utilities_bit_hpp,
                        cudf::experimental::rolling::jit::code::operation_h },
                      { "-std=c++14", "-w" }, nullptr, stream)
    .set_kernel_inst("gpu_rolling_new", // name of the kernel we are launching
                      { cudf::jit::get_type_name(input.type()), // list of template arguments
                        cudf::jit::get_type_name(output->type()),
                        udf_agg->_operator_name,
                        static_window ? "cudf::size_type" : "cudf::size_type*"})
    .launch(input.size(), cudf::jit::get_data_ptr(input), input.null_mask(),
            cudf::jit::get_data_ptr(output_view), output_view.null_mask(),
            device_valid_count.data(), preceding_window, following_window, min_periods);

  output->set_null_count(output->size() - device_valid_count.value(stream));

  // check the stream for debugging
  CHECK_CUDA(stream);

  cudf::nvtx::range_pop();

  return output;
}

// Applies a rolling window function to the values in a column.
template <typename WindowIterator>
std::unique_ptr<column> rolling_window(column_view const& input,
                                       WindowIterator preceding_window_begin,
                                       WindowIterator following_window_begin,
                                       size_type min_periods,
                                       std::unique_ptr<aggregation> const& agg,
                                       rmm::mr::device_memory_resource* mr,
                                       hipStream_t stream = 0)
{
  static_assert(warp_size == cudf::detail::size_in_bits<cudf::bitmask_type>(),
                "bitmask_type size does not match CUDA warp size");

  return cudf::experimental::type_dispatcher(input.type(),
                                             dispatch_rolling{},
                                             input,
                                             preceding_window_begin,
                                             following_window_begin,
                                             min_periods, agg, mr, stream);

}

} // namespace detail

// Applies a fixed-size rolling window function to the values in a column.
std::unique_ptr<column> rolling_window(column_view const& input,
                                       size_type preceding_window,
                                       size_type following_window,
                                       size_type min_periods,
                                       std::unique_ptr<aggregation> const& agg,
                                       rmm::mr::device_memory_resource* mr)
{
  CUDF_EXPECTS((min_periods >= 0), "min_periods must be non-negative");

  if (agg->kind == aggregation::CUDA || agg->kind == aggregation::PTX) {
    return cudf::experimental::detail::rolling_window_udf<true>(input,
                                                                preceding_window,
                                                                following_window,
                                                                min_periods, agg, mr, 0);
  } else {
    auto preceding_window_begin = thrust::make_constant_iterator(preceding_window);
    auto following_window_begin = thrust::make_constant_iterator(following_window);

    return cudf::experimental::detail::rolling_window(input,
                                                      preceding_window_begin,
                                                      following_window_begin,
                                                      min_periods, agg, mr, 0);
  }
}

// Applies a variable-size rolling window function to the values in a column.
std::unique_ptr<column> rolling_window(column_view const& input,
                                       column_view const& preceding_window,
                                       column_view const& following_window,
                                       size_type min_periods,
                                       std::unique_ptr<aggregation> const& agg,
                                       rmm::mr::device_memory_resource* mr)
{
  if (preceding_window.size() == 0 || following_window.size() == 0) return empty_like(input);

  CUDF_EXPECTS(preceding_window.type().id() == INT32 && following_window.type().id() == INT32,
               "preceding_window/following_window must have INT32 type");

  CUDF_EXPECTS(preceding_window.size() == input.size() && following_window.size() == input.size(),
               "preceding_window/following_window size must match input size");

  if (agg->kind == aggregation::CUDA || agg->kind == aggregation::PTX) {
    return cudf::experimental::detail::rolling_window_udf<false>(input,
                                                                 preceding_window.begin<size_type>(),
                                                                 following_window.begin<size_type>(),
                                                                 min_periods, agg, mr, 0);
  } else {
    return cudf::experimental::detail::rolling_window(input, 
                                                      preceding_window.begin<size_type>(),
                                                      following_window.begin<size_type>(),
                                                      min_periods, agg, mr, 0);
  }
}

} // namespace experimental 
} // namespace cudf
