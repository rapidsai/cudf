#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019-2020, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <cudf/types.hpp>
#include <cudf/column/column_view.hpp>
#include <cudf/column/column_device_view.cuh>
#include <cudf/column/column_factories.hpp>
#include <cudf/detail/aggregation/aggregation.cuh>
#include <cudf/aggregation.hpp>
#include <cudf/detail/gather.hpp>
#include <cudf/utilities/nvtx_utils.hpp>
#include <cudf/utilities/bit.hpp>
#include <cudf/detail/copy.hpp>
#include <cudf/detail/utilities/cuda.cuh>
#include <cudf/copying.hpp>
#include <rolling/rolling_detail.hpp>
#include <cudf/rolling.hpp>
#include <cudf/detail/nvtx/ranges.hpp>

#include <jit/type.h>
#include <jit/launcher.h>
#include <jit/parser.h>
#include <rolling/jit/code/code.h>

#include <types.hpp.jit>
#include <bit.hpp.jit>

#include <rmm/device_scalar.hpp>

#include <memory>

namespace cudf {
namespace experimental {

namespace detail {

namespace { // anonymous
/**
 * @brief Only count operation is executed and count is updated
 *        depending on `min_periods` and returns true if it was
 *        valid, else false.
 */
template <typename InputType, typename OutputType, typename agg_op, aggregation::Kind op, bool has_nulls>
std::enable_if_t<op == aggregation::COUNT_VALID || op == aggregation::COUNT_ALL, bool>
__device__
process_rolling_window(column_device_view input,
                        mutable_column_device_view output,
                        size_type start_index,
                        size_type end_index,
                        size_type current_index,
                        size_type min_periods,
                        InputType identity) {

    // declare this as volatile to avoid some compiler optimizations that lead to incorrect results
    // for CUDA 10.0 and below (fixed in CUDA 10.1)
    volatile cudf::size_type count = 0;
    
    for (size_type j = start_index; j < end_index; j++) {
        if (op == aggregation::COUNT_ALL || !has_nulls || input.is_valid(j)) {
            count++;
        }
    }
   
    bool output_is_valid = (count >= min_periods);
    output.element<OutputType>(current_index) = count;

    return output_is_valid;
}

/**
 * @brief Only used for `string_view` type to get ARGMIN and ARGMAX, which
 *        will be used to gather MIN and MAX. And returns true if the
 *        operation was valid, else false.
 */
template <typename InputType, typename OutputType, typename agg_op, aggregation::Kind op, bool has_nulls>
std::enable_if_t<(op == aggregation::ARGMIN  or op == aggregation::ARGMAX) and
                 std::is_same<InputType, cudf::string_view>::value, bool>
__device__
process_rolling_window(column_device_view input,
                        mutable_column_device_view output,
                        size_type start_index,
                        size_type end_index,
                        size_type current_index,
                        size_type min_periods,
                        InputType identity) {

    // declare this as volatile to avoid some compiler optimizations that lead to incorrect results
    // for CUDA 10.0 and below (fixed in CUDA 10.1)
    volatile cudf::size_type count = 0;
    InputType val = identity;
    OutputType val_index = (op == aggregation::ARGMIN)? ARGMIN_SENTINEL : ARGMAX_SENTINEL;

    for (size_type j = start_index; j < end_index; j++) {
        if (!has_nulls || input.is_valid(j)) {
            InputType element = input.element<InputType>(j);
            val = agg_op{}(element, val);
            if (val == element) {
                val_index = j;
            }
            count++;
        }
    }

    bool output_is_valid = (count >= min_periods);
    // -1 will help identify null elements while gathering for Min and Max
    // In case of count, this would be null, so doesn't matter.
    output.element<OutputType>(current_index) = (output_is_valid)? val_index : -1;

    // The gather mask shouldn't contain null values, so
    // always return zero
    return true;
}

/**
 * @brief Operates on only fixed-width types and returns true if the
 *        operation was valid, else false.
 */
template <typename InputType, typename OutputType, typename agg_op, aggregation::Kind op, bool has_nulls>
std::enable_if_t<!std::is_same<InputType, cudf::string_view>::value and
                 !(op == aggregation::COUNT_VALID || op == aggregation::COUNT_ALL), bool>
__device__
process_rolling_window(column_device_view input,
                        mutable_column_device_view output,
                        size_type start_index,
                        size_type end_index,
                        size_type current_index,
                        size_type min_periods,
                        InputType identity) {

    // declare this as volatile to avoid some compiler optimizations that lead to incorrect results
    // for CUDA 10.0 and below (fixed in CUDA 10.1)
    volatile cudf::size_type count = 0;
    OutputType val = agg_op::template identity<OutputType>();

    for (size_type j = start_index; j < end_index; j++) {
        if (!has_nulls || input.is_valid(j)) {
            OutputType element = input.element<InputType>(j);
            val = agg_op{}(element, val);
            count++;
        }
    }

    bool output_is_valid = (count >= min_periods);

    // store the output value, one per thread
    cudf::detail::store_output_functor<OutputType, op == aggregation::MEAN>{}(output.element<OutputType>(current_index),
                val, count);

    return output_is_valid;
}

/**
 * @brief Computes the rolling window function
 *
 * @tparam InputType  Datatype of `input`
 * @tparam OutputType  Datatype of `output`
 * @tparam agg_op  A functor that defines the aggregation operation
 * @tparam op The aggregation operator (enum value)
 * @tparam block_size CUDA block size for the kernel
 * @tparam has_nulls true if the input column has nulls
 * @tparam WindowIterator iterator type (inferred)
 * @param input Input column device view
 * @param output Output column device view
 * @param preceding_window_begin[in] Rolling window size iterator, accumulates from
 *                in_col[i-preceding_window] to in_col[i] inclusive
 * @param following_window_begin[in] Rolling window size iterator in the forward
 *                direction, accumulates from in_col[i] to
 *                in_col[i+following_window] inclusive
 * @param min_periods[in]  Minimum number of observations in window required to
 *                have a value, otherwise 0 is stored in the valid bit mask
 * @param identity identity value of `InputType`
 */
template <typename InputType, typename OutputType, typename agg_op, aggregation::Kind op, 
         int block_size, bool has_nulls, typename WindowIterator>
__launch_bounds__(block_size)
__global__
void gpu_rolling(column_device_view input,
                 mutable_column_device_view output,
                 size_type * __restrict__ output_valid_count,
                 WindowIterator preceding_window_begin,
                 WindowIterator following_window_begin,
                 size_type min_periods,
                 InputType identity)
{
  size_type i = blockIdx.x * block_size + threadIdx.x;
  size_type stride = block_size * gridDim.x;

  size_type warp_valid_count{0};

  auto active_threads = __ballot_sync(0xffffffff, i < input.size());
  while(i < input.size())
  {

    size_type preceding_window = preceding_window_begin[i];
    size_type following_window = following_window_begin[i];

    // compute bounds
    size_type start = min(input.size(), max(0, i - preceding_window + 1));
    size_type end = min(input.size(), max(0, i + following_window + 1));
    size_type start_index = min(start, end);
    size_type end_index = max(start, end);

    // aggregate
    // TODO: We should explore using shared memory to avoid redundant loads.
    //       This might require separating the kernel into a special version
    //       for dynamic and static sizes.

    volatile bool output_is_valid = false;
    output_is_valid = process_rolling_window<InputType, OutputType, agg_op,
                           op, has_nulls>(input, output, start_index, end_index, i, min_periods, identity); 

    // set the mask
    cudf::bitmask_type result_mask{__ballot_sync(active_threads, output_is_valid)};

    // only one thread writes the mask
    if (0 == threadIdx.x % cudf::experimental::detail::warp_size) {
      output.set_mask_word(cudf::word_index(i), result_mask);
      warp_valid_count += __popc(result_mask);
    }

    // process next element 
    i += stride;
    active_threads = __ballot_sync(active_threads, i < input.size());
  }

  // sum the valid counts across the whole block  
  size_type block_valid_count = 
    cudf::experimental::detail::single_lane_block_sum_reduce<block_size, 0>(warp_valid_count);

  if(threadIdx.x == 0) {
    atomicAdd(output_valid_count, block_valid_count);
  }
}

template <typename InputType>
struct rolling_window_launcher
{

  template <typename T, typename agg_op, aggregation::Kind op, typename WindowIterator>
  size_type kernel_launcher(column_view const& input,
                       mutable_column_view& output,
                       WindowIterator preceding_window_begin,
                       WindowIterator following_window_begin,
                       size_type min_periods,
                       std::unique_ptr<aggregation> const& agg,
                       T identity,
                       hipStream_t stream) {
      cudf::nvtx::range_push("CUDF_ROLLING_WINDOW", cudf::nvtx::color::ORANGE);

      constexpr cudf::size_type block_size = 256;
      cudf::experimental::detail::grid_1d grid(input.size(), block_size);

      auto input_device_view = column_device_view::create(input, stream);
      auto output_device_view = mutable_column_device_view::create(output, stream);

      rmm::device_scalar<size_type> device_valid_count{0, stream};

      if (input.has_nulls()) {
          gpu_rolling<T, target_type_t<InputType, op>, agg_op, op, block_size, true><<<grid.num_blocks, block_size, 0, stream>>>
              (*input_device_view, *output_device_view, device_valid_count.data(),
               preceding_window_begin, following_window_begin, min_periods, identity);
      } else {
          gpu_rolling<T, target_type_t<InputType, op>, agg_op, op, block_size, false><<<grid.num_blocks, block_size, 0, stream>>>
              (*input_device_view, *output_device_view, device_valid_count.data(),
               preceding_window_begin, following_window_begin, min_periods, identity);
      }

      size_type valid_count = device_valid_count.value(stream);

      // check the stream for debugging
      CHECK_CUDA(stream);
      
      cudf::nvtx::range_pop();

      return valid_count;
  }

  // This launch is only for fixed width columns with valid aggregation option
  // numeric: All
  // timestamp: MIN, MAX, COUNT_VALID, COUNT_ALL
  template <typename T, typename agg_op, aggregation::Kind op, typename WindowIterator>
  std::enable_if_t<(cudf::detail::is_supported<T, agg_op,
                                  op, op == aggregation::MEAN>()) and
                   !(cudf::detail::is_string_supported<T, agg_op, op>()), std::unique_ptr<column>>
  launch(column_view const& input,
         WindowIterator preceding_window_begin,
         WindowIterator following_window_begin,
         size_type min_periods,
         std::unique_ptr<aggregation> const& agg,
         rmm::mr::device_memory_resource *mr,
         hipStream_t stream) {

      if (input.is_empty()) return empty_like(input);

      auto output = make_fixed_width_column(target_type(input.type(), op), input.size(),
              mask_state::UNINITIALIZED, stream, mr);

      cudf::mutable_column_view output_view = output->mutable_view();
      auto valid_count = kernel_launcher<T, agg_op, op, WindowIterator>(input, output_view, preceding_window_begin,
              following_window_begin, min_periods, agg, agg_op::template identity<T>(), stream);

      output->set_null_count(output->size() - valid_count);

      return output;
  }

  // This launch is only for string columns with valid aggregation option
  // string: MIN, MAX, COUNT_VALID, COUNT_ALL
  template <typename T, typename agg_op, aggregation::Kind op, typename WindowIterator>
  std::enable_if_t<!(cudf::detail::is_supported<T, agg_op,
                                  op, op == aggregation::MEAN>()) and
                   (cudf::detail::is_string_supported<T, agg_op, op>()), std::unique_ptr<column>>
  launch(column_view const& input,
         WindowIterator preceding_window_begin,
         WindowIterator following_window_begin,
         size_type min_periods,
         std::unique_ptr<aggregation> const& agg,
         rmm::mr::device_memory_resource *mr,
         hipStream_t stream) {

      if (input.is_empty()) return empty_like(input);

      auto output = make_numeric_column(cudf::data_type{cudf::experimental::type_to_id<size_type>()},
            input.size(), cudf::mask_state::UNINITIALIZED, stream, mr);

      cudf::mutable_column_view output_view = output->mutable_view();

      // Passing the agg_op and aggregation::Kind as constant to group them in pair, else it
      // evolves to error when try to use agg_op as compiler tries different combinations
      if(op == aggregation::MIN) {
          kernel_launcher<T, DeviceMin, aggregation::ARGMIN, WindowIterator>(input, output_view, preceding_window_begin,
                  following_window_begin, min_periods, agg, DeviceMin::template identity<T>(), stream);
      } else if(op == aggregation::MAX) {
          kernel_launcher<T, DeviceMax, aggregation::ARGMAX, WindowIterator>(input, output_view, preceding_window_begin,
                  following_window_begin, min_periods, agg, DeviceMax::template identity<T>(), stream);
      } else {
          CUDF_EXPECTS(op == aggregation::COUNT_VALID || 
                       op == aggregation::COUNT_ALL,
                       "COUNT_VALID or COUNT_ALL aggregation only is expected");
          size_type valid_count;
          if (op == aggregation::COUNT_ALL)
            valid_count = kernel_launcher<T, DeviceCount, aggregation::COUNT_ALL, WindowIterator>(input, output_view, preceding_window_begin,
                  following_window_begin, min_periods, agg, string_view{}, stream);
          else 
            valid_count = kernel_launcher<T, DeviceCount, aggregation::COUNT_VALID, WindowIterator>(input, output_view, preceding_window_begin,
                  following_window_begin, min_periods, agg, string_view{}, stream);
          output->set_null_count(output->size() - valid_count);
      }

      // If aggregation operation is MIN or MAX, then the output we got is a gather map
      if((op == aggregation::MIN) or (op == aggregation::MAX)) {
          // The rows that represent null elements will be having negative values in gather map,
          // and that's why nullify_out_of_bounds/ignore_out_of_bounds is true.
          auto output_table = detail::gather(table_view{{input}}, output->view(), false, true, false, mr, stream);
          return std::make_unique<cudf::column>(std::move(output_table->get_column(0)));;
      }

      return output;
  }

  // Deals with invalid column and/or aggregation options
  template <typename T, typename agg_op, aggregation::Kind op, typename WindowIterator>
  std::enable_if_t<!(cudf::detail::is_supported<T, agg_op,
                                  op, op == aggregation::MEAN>()) and
                   !(cudf::detail::is_string_supported<T, agg_op, op>()), std::unique_ptr<column>>
  launch(column_view const& input,
         WindowIterator preceding_window_begin,
         WindowIterator following_window_begin,
         size_type min_periods,
         std::unique_ptr<aggregation> const& agg,
         rmm::mr::device_memory_resource *mr,
         hipStream_t stream) {

      CUDF_FAIL("Aggregation operator and/or input type combination is invalid");
  }


  template<aggregation::Kind op, typename WindowIterator>
  std::enable_if_t<!(op == aggregation::MEAN), std::unique_ptr<column>>
  operator()(column_view const& input,
                                     WindowIterator preceding_window_begin,
                                     WindowIterator following_window_begin,
                                     size_type min_periods,
                                     std::unique_ptr<aggregation> const& agg,
                                     rmm::mr::device_memory_resource *mr,
                                     hipStream_t stream)
  {
      return launch <InputType, typename corresponding_operator<op>::type, op, WindowIterator> (
              input,
              preceding_window_begin,
              following_window_begin,
              min_periods,
              agg,
              mr,
              stream);
  }

  // This variant is just to handle mean
  template<aggregation::Kind op, typename WindowIterator>
  std::enable_if_t<(op == aggregation::MEAN), std::unique_ptr<column>>
  operator()(column_view const& input,
                                     WindowIterator preceding_window_begin,
                                     WindowIterator following_window_begin,
                                     size_type min_periods,
                                     std::unique_ptr<aggregation> const& agg,
                                     rmm::mr::device_memory_resource *mr,
                                     hipStream_t stream) {

      return launch <InputType, cudf::DeviceSum, op, WindowIterator> (
              input,
              preceding_window_begin,
              following_window_begin,
              min_periods,
              agg,
              mr,
              stream);
  }


};

struct dispatch_rolling {
    template <typename T, typename WindowIterator>
    std::unique_ptr<column> operator()(column_view const& input,
                                     WindowIterator preceding_window_begin,
                                     WindowIterator following_window_begin,
                                     size_type min_periods,
                                     std::unique_ptr<aggregation> const& agg,
                                     rmm::mr::device_memory_resource *mr,
                                     hipStream_t stream) {

        return aggregation_dispatcher(agg->kind, rolling_window_launcher<T>{},
                                      input,
                                      preceding_window_begin, following_window_begin,
                                      min_periods, agg, mr, stream);
    }
};

} // namespace anonymous

// Applies a user-defined rolling window function to the values in a column.
template <bool static_window, typename WindowIterator>
std::unique_ptr<column> rolling_window_udf(column_view const &input,
                                           WindowIterator preceding_window,
                                           WindowIterator following_window,
                                           size_type min_periods,
                                           std::unique_ptr<aggregation> const& agg,
                                           rmm::mr::device_memory_resource* mr,
                                           hipStream_t stream = 0)
{
  static_assert(warp_size == cudf::detail::size_in_bits<cudf::bitmask_type>(),
                "bitmask_type size does not match CUDA warp size");

  if (input.has_nulls())
    CUDF_FAIL("Currently the UDF version of rolling window does NOT support inputs with nulls.");

  cudf::nvtx::range_push("CUDF_ROLLING_WINDOW", cudf::nvtx::color::ORANGE);

  min_periods = std::max(min_periods, 0);

  auto udf_agg = static_cast<udf_aggregation*>(agg.get());

  std::string hash = "prog_experimental_rolling." 
    + std::to_string(std::hash<std::string>{}(udf_agg->_source));
  
  std::string cuda_source;
  switch(udf_agg->kind){
    case aggregation::Kind::PTX:
      cuda_source = cudf::experimental::rolling::jit::code::kernel_headers;
      cuda_source += cudf::jit::parse_single_function_ptx(udf_agg->_source, udf_agg->_function_name,
                                                          cudf::jit::get_type_name(udf_agg->_output_type),
                                                          {0, 5}); // args 0 and 5 are pointers.
      cuda_source += cudf::experimental::rolling::jit::code::kernel;
      break; 
    case aggregation::Kind::CUDA:
      cuda_source = cudf::experimental::rolling::jit::code::kernel_headers;
      cuda_source += cudf::jit::parse_single_function_cuda(udf_agg->_source, udf_agg->_function_name);
      cuda_source += cudf::experimental::rolling::jit::code::kernel;
      break;
    default:
      CUDF_FAIL("Unsupported UDF type.");
  }

  std::unique_ptr<column> output = make_numeric_column(udf_agg->_output_type, input.size(),
                                                       cudf::mask_state::UNINITIALIZED, stream, mr);

  auto output_view = output->mutable_view();
  rmm::device_scalar<size_type> device_valid_count{0, stream};

  const std::vector<std::string> compiler_flags{
    "-std=c++14",
    // Have jitify prune unused global variables
    "-remove-unused-globals",
    // suppress all NVRTC warnings
    "-w"
  };

  // Launch the jitify kernel
  cudf::jit::launcher(hash, cuda_source,
                      { cudf_types_hpp, cudf_utilities_bit_hpp,
                        cudf::experimental::rolling::jit::code::operation_h },
                      compiler_flags, nullptr, stream)
    .set_kernel_inst("gpu_rolling_new", // name of the kernel we are launching
                      { cudf::jit::get_type_name(input.type()), // list of template arguments
                        cudf::jit::get_type_name(output->type()),
                        udf_agg->_operator_name,
                        static_window ? "cudf::size_type" : "cudf::size_type*"})
    .launch(input.size(), cudf::jit::get_data_ptr(input), input.null_mask(),
            cudf::jit::get_data_ptr(output_view), output_view.null_mask(),
            device_valid_count.data(), preceding_window, following_window, min_periods);

  output->set_null_count(output->size() - device_valid_count.value(stream));

  // check the stream for debugging
  CHECK_CUDA(stream);

  cudf::nvtx::range_pop();

  return output;
}

/**
* @copydoc cudf::experimental::rolling_window(
*                                  column_view const& input,
*                                  WindowIterator preceding_window_begin,
*                                  WindowIterator following_window_begin,
*                                  size_type min_periods,
*                                  std::unique_ptr<aggregation> const& agg,
*                                  rmm::mr::device_memory_resource* mr)
*
* @param stream The stream to use for CUDA operations
*/
template <typename WindowIterator>
std::unique_ptr<column> rolling_window(column_view const& input,
                                       WindowIterator preceding_window_begin,
                                       WindowIterator following_window_begin,
                                       size_type min_periods,
                                       std::unique_ptr<aggregation> const& agg,
                                       rmm::mr::device_memory_resource* mr,
                                       hipStream_t stream = 0)
{
  static_assert(warp_size == cudf::detail::size_in_bits<cudf::bitmask_type>(),
                "bitmask_type size does not match CUDA warp size");

  min_periods = std::max(min_periods, 0);

  return cudf::experimental::type_dispatcher(input.type(),
                                             dispatch_rolling{},
                                             input,
                                             preceding_window_begin,
                                             following_window_begin,
                                             min_periods, agg, mr, stream);

}

} // namespace detail

// Applies a fixed-size rolling window function to the values in a column.
std::unique_ptr<column> rolling_window(column_view const& input,
                                       size_type preceding_window,
                                       size_type following_window,
                                       size_type min_periods,
                                       std::unique_ptr<aggregation> const& agg,
                                       rmm::mr::device_memory_resource* mr)
{
  CUDF_FUNC_RANGE();
  if (input.size() == 0) return empty_like(input);
  CUDF_EXPECTS((min_periods >= 0), "min_periods must be non-negative");

  if (agg->kind == aggregation::CUDA || agg->kind == aggregation::PTX) {
    return cudf::experimental::detail::rolling_window_udf<true>(input,
                                                                preceding_window,
                                                                following_window,
                                                                min_periods, agg, mr, 0);
  } else {
    auto preceding_window_begin = thrust::make_constant_iterator(preceding_window);
    auto following_window_begin = thrust::make_constant_iterator(following_window);

    return cudf::experimental::detail::rolling_window(input,
                                                      preceding_window_begin,
                                                      following_window_begin,
                                                      min_periods, agg, mr, 0);
  }
}

// Applies a variable-size rolling window function to the values in a column.
std::unique_ptr<column> rolling_window(column_view const& input,
                                       column_view const& preceding_window,
                                       column_view const& following_window,
                                       size_type min_periods,
                                       std::unique_ptr<aggregation> const& agg,
                                       rmm::mr::device_memory_resource* mr)
{
  CUDF_FUNC_RANGE();
  if (preceding_window.size() == 0 || following_window.size() == 0 || input.size() == 0) return empty_like(input);

  CUDF_EXPECTS(preceding_window.type().id() == INT32 && following_window.type().id() == INT32,
               "preceding_window/following_window must have INT32 type");

  CUDF_EXPECTS(preceding_window.size() == input.size() && following_window.size() == input.size(),
               "preceding_window/following_window size must match input size");

  if (agg->kind == aggregation::CUDA || agg->kind == aggregation::PTX) {
    return cudf::experimental::detail::rolling_window_udf<false>(input,
                                                                 preceding_window.begin<size_type>(),
                                                                 following_window.begin<size_type>(),
                                                                 min_periods, agg, mr, 0);
  } else {
    return cudf::experimental::detail::rolling_window(input, 
                                                      preceding_window.begin<size_type>(),
                                                      following_window.begin<size_type>(),
                                                      min_periods, agg, mr, 0);
  }
}

} // namespace experimental 
} // namespace cudf
