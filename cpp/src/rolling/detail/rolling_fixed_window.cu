#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2022, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "rolling.cuh"

#include <cudf/detail/aggregation/aggregation.hpp>
#include <cudf/utilities/default_stream.hpp>

#include <thrust/extrema.h>
#include <thrust/iterator/constant_iterator.h>

namespace cudf::detail {

// Applies a fixed-size rolling window function to the values in a column.
std::unique_ptr<column> rolling_window(column_view const& input,
                                       column_view const& default_outputs,
                                       size_type preceding_window,
                                       size_type following_window,
                                       size_type min_periods,
                                       rolling_aggregation const& agg,
                                       rmm::cuda_stream_view stream,
                                       rmm::mr::device_memory_resource* mr)
{
  CUDF_FUNC_RANGE();

  if (input.is_empty()) { return cudf::detail::empty_output_for_rolling_aggregation(input, agg); }

  CUDF_EXPECTS((min_periods >= 0), "min_periods must be non-negative");

  CUDF_EXPECTS((default_outputs.is_empty() || default_outputs.size() == input.size()),
               "Defaults column must be either empty or have as many rows as the input column.");

  if (agg.kind == aggregation::CUDA || agg.kind == aggregation::PTX) {
    // TODO: In future, might need to clamp preceding/following to column boundaries.
    return cudf::detail::rolling_window_udf(input,
                                            preceding_window,
                                            "cudf::size_type",
                                            following_window,
                                            "cudf::size_type",
                                            min_periods,
                                            agg,
                                            stream,
                                            mr);
  } else {
    // Clamp preceding/following to column boundaries.
    // E.g. If preceding_window == 2, then for a column of 5 elements, preceding_window will be:
    //      [1, 2, 2, 2, 1]
    auto const preceding_window_begin = cudf::detail::make_counting_transform_iterator(
      0,
      [preceding_window] __device__(size_type i) { return thrust::min(i + 1, preceding_window); });
    auto const following_window_begin = cudf::detail::make_counting_transform_iterator(
      0, [col_size = input.size(), following_window] __device__(size_type i) {
        return thrust::min(col_size - i - 1, following_window);
      });

    return cudf::detail::rolling_window(input,
                                        default_outputs,
                                        preceding_window_begin,
                                        following_window_begin,
                                        min_periods,
                                        agg,
                                        stream,
                                        mr);
  }
}
}  // namespace cudf::detail
