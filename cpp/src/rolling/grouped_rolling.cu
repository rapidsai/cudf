#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2020-2021, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "range_window_bounds_detail.hpp"
#include "rolling_detail.cuh"
#include "rolling_jit_detail.hpp"

#include <cudf/detail/iterator.cuh>
#include <cudf/detail/rolling.hpp>
#include <cudf/detail/utilities/vector_factories.hpp>
#include <cudf/rolling/range_window_bounds.hpp>
#include <cudf/types.hpp>
#include <cudf/unary.hpp>

namespace cudf {
std::unique_ptr<column> grouped_rolling_window(table_view const& group_keys,
                                               column_view const& input,
                                               size_type preceding_window,
                                               size_type following_window,
                                               size_type min_periods,
                                               rolling_aggregation const& aggr,
                                               rmm::mr::device_memory_resource* mr)
{
  return grouped_rolling_window(group_keys,
                                input,
                                window_bounds::get(preceding_window),
                                window_bounds::get(following_window),
                                min_periods,
                                aggr,
                                mr);
}

std::unique_ptr<column> grouped_rolling_window(table_view const& group_keys,
                                               column_view const& input,
                                               window_bounds preceding_window,
                                               window_bounds following_window,
                                               size_type min_periods,
                                               rolling_aggregation const& aggr,
                                               rmm::mr::device_memory_resource* mr)
{
  return grouped_rolling_window(group_keys,
                                input,
                                empty_like(input)->view(),
                                preceding_window,
                                following_window,
                                min_periods,
                                aggr,
                                mr);
}

std::unique_ptr<column> grouped_rolling_window(table_view const& group_keys,
                                               column_view const& input,
                                               column_view const& default_outputs,
                                               size_type preceding_window,
                                               size_type following_window,
                                               size_type min_periods,
                                               rolling_aggregation const& aggr,
                                               rmm::mr::device_memory_resource* mr)
{
  return grouped_rolling_window(group_keys,
                                input,
                                default_outputs,
                                window_bounds::get(preceding_window),
                                window_bounds::get(following_window),
                                min_periods,
                                aggr,
                                mr);
}

namespace detail {

std::unique_ptr<column> grouped_rolling_window(table_view const& group_keys,
                                               column_view const& input,
                                               column_view const& default_outputs,
                                               window_bounds preceding_window_bounds,
                                               window_bounds following_window_bounds,
                                               size_type min_periods,
                                               rolling_aggregation const& aggr,
                                               rmm::cuda_stream_view stream,
                                               rmm::mr::device_memory_resource* mr)
{
  CUDF_FUNC_RANGE();

  if (input.is_empty()) { return cudf::detail::empty_output_for_rolling_aggregation(input, aggr); }

  CUDF_EXPECTS((group_keys.num_columns() == 0 || group_keys.num_rows() == input.size()),
               "Size mismatch between group_keys and input vector.");

  CUDF_EXPECTS((min_periods > 0), "min_periods must be positive");

  CUDF_EXPECTS((default_outputs.is_empty() || default_outputs.size() == input.size()),
               "Defaults column must be either empty or have as many rows as the input column.");

  auto const preceding_window = preceding_window_bounds.value;
  auto const following_window = following_window_bounds.value;

  if (group_keys.num_columns() == 0) {
    // No Groupby columns specified. Treat as one big group.
    return rolling_window(
      input, default_outputs, preceding_window, following_window, min_periods, aggr, mr);
  }

  using sort_groupby_helper = cudf::groupby::detail::sort::sort_groupby_helper;

  sort_groupby_helper helper{group_keys, cudf::null_policy::INCLUDE, cudf::sorted::YES};
  auto const& group_offsets{helper.group_offsets(stream)};
  auto const& group_labels{helper.group_labels(stream)};

  // `group_offsets` are interpreted in adjacent pairs, each pair representing the offsets
  // of the first, and one past the last elements in a group.
  //
  // If `group_offsets` is not empty, it must contain at least two offsets:
  //   a. 0, indicating the first element in `input`
  //   b. input.size(), indicating one past the last element in `input`.
  //
  // Thus, for an input of 1000 rows,
  //   0. [] indicates a single group, spanning the entire column.
  //   1  [10] is invalid.
  //   2. [0, 1000] indicates a single group, spanning the entire column (thus, equivalent to no
  //   groups.)
  //   3. [0, 500, 1000] indicates two equal-sized groups: [0,500), and [500,1000).

  assert(group_offsets.size() >= 2 && group_offsets.element(0, stream) == 0 &&
         group_offsets.element(group_offsets.size() - 1, stream) == input.size() &&
         "Must have at least one group.");

  auto preceding_calculator = [d_group_offsets = group_offsets.data(),
                               d_group_labels  = group_labels.data(),
                               preceding_window] __device__(size_type idx) {
    auto group_label = d_group_labels[idx];
    auto group_start = d_group_offsets[group_label];
    return thrust::minimum<size_type>{}(preceding_window,
                                        idx - group_start + 1);  // Preceding includes current row.
  };

  auto following_calculator = [d_group_offsets = group_offsets.data(),
                               d_group_labels  = group_labels.data(),
                               following_window] __device__(size_type idx) {
    auto group_label = d_group_labels[idx];
    auto group_end   = d_group_offsets[group_label + 1];  // Cannot fall off the end, since offsets
                                                          // is capped with `input.size()`.
    return thrust::minimum<size_type>{}(following_window, (group_end - 1) - idx);
  };

  if (aggr.kind == aggregation::CUDA || aggr.kind == aggregation::PTX) {
    cudf::detail::preceding_window_wrapper grouped_preceding_window{
      group_offsets.data(), group_labels.data(), preceding_window};

    cudf::detail::following_window_wrapper grouped_following_window{
      group_offsets.data(), group_labels.data(), following_window};

    return cudf::detail::rolling_window_udf(input,
                                            grouped_preceding_window,
                                            "cudf::detail::preceding_window_wrapper",
                                            grouped_following_window,
                                            "cudf::detail::following_window_wrapper",
                                            min_periods,
                                            aggr,
                                            stream,
                                            mr);
  } else {
    return cudf::detail::rolling_window(
      input,
      default_outputs,
      cudf::detail::make_counting_transform_iterator(0, preceding_calculator),
      cudf::detail::make_counting_transform_iterator(0, following_calculator),
      min_periods,
      aggr,
      stream,
      mr);
  }
}

}  // namespace detail

std::unique_ptr<column> grouped_rolling_window(table_view const& group_keys,
                                               column_view const& input,
                                               column_view const& default_outputs,
                                               window_bounds preceding_window_bounds,
                                               window_bounds following_window_bounds,
                                               size_type min_periods,
                                               rolling_aggregation const& aggr,
                                               rmm::mr::device_memory_resource* mr)
{
  return detail::grouped_rolling_window(group_keys,
                                        input,
                                        default_outputs,
                                        preceding_window_bounds,
                                        following_window_bounds,
                                        min_periods,
                                        aggr,
                                        rmm::cuda_stream_default,
                                        mr);
}

namespace {

/// For order-by columns of signed types, bounds calculation might cause accidental
/// overflow/underflows. This needs to be detected and handled appropriately
/// for signed and unsigned types.

/**
 * @brief Add `delta` to value, and cap at numeric_limits::max(), for signed types.
 */
template <typename T, std::enable_if_t<std::numeric_limits<T>::is_signed>* = nullptr>
__device__ T add_safe(T const& value, T const& delta)
{
  // delta >= 0.
  return (value < 0 || (std::numeric_limits<T>::max() - value) >= delta)
           ? (value + delta)
           : std::numeric_limits<T>::max();
}

/**
 * @brief Add `delta` to value, and cap at numeric_limits::max(), for unsigned types.
 */
template <typename T, std::enable_if_t<!std::numeric_limits<T>::is_signed>* = nullptr>
__device__ T add_safe(T const& value, T const& delta)
{
  // delta >= 0.
  return ((std::numeric_limits<T>::max() - value) >= delta) ? (value + delta)
                                                            : std::numeric_limits<T>::max();
}

/**
 * @brief Subtract `delta` from value, and cap at numeric_limits::min(), for signed types.
 */
template <typename T, std::enable_if_t<std::numeric_limits<T>::is_signed>* = nullptr>
__device__ T subtract_safe(T const& value, T const& delta)
{
  // delta >= 0;
  return (value >= 0 || (value - std::numeric_limits<T>::min()) >= delta)
           ? (value - delta)
           : std::numeric_limits<T>::min();
}

/**
 * @brief Subtract `delta` from value, and cap at numeric_limits::min(), for unsigned types.
 */
template <typename T, std::enable_if_t<!std::numeric_limits<T>::is_signed>* = nullptr>
__device__ T subtract_safe(T const& value, T const& delta)
{
  // delta >= 0;
  return ((value - std::numeric_limits<T>::min()) >= delta) ? (value - delta)
                                                            : std::numeric_limits<T>::min();
}

/// Given a single, ungrouped order-by column, return the indices corresponding
/// to the first null element, and (one past) the last null timestamp.
/// The input column is sorted, with all null values clustered either
/// at the beginning of the column or at the end.
/// If no null values are founds, null_begin and null_end are 0.
std::tuple<size_type, size_type> get_null_bounds_for_orderby_column(
  column_view const& orderby_column)
{
  auto const num_rows  = orderby_column.size();
  auto const num_nulls = orderby_column.null_count();

  if (num_nulls == num_rows || num_nulls == 0) {
    // Short-circuit: All nulls, or no nulls.
    return std::make_tuple(0, num_nulls);
  }

  auto const first_row_is_null = orderby_column.null_count(0, 1) == 1;

  return first_row_is_null ? std::make_tuple(0, num_nulls)
                           : std::make_tuple(num_rows - num_nulls, num_rows);
}

template <typename Calculator>
std::unique_ptr<column> expand_to_column(Calculator const& calc,
                                         size_type const& num_rows,
                                         rmm::cuda_stream_view stream,
                                         rmm::mr::device_memory_resource* mr)
{
  auto window_column = cudf::make_fixed_width_column(
    cudf::data_type{type_id::INT32}, num_rows, cudf::mask_state::UNALLOCATED, stream, mr);

  auto begin = cudf::detail::make_counting_transform_iterator(0, calc);

  thrust::copy_n(
    rmm::exec_policy(stream), begin, num_rows, window_column->mutable_view().data<size_type>());

  return window_column;
}

/// Range window computation, with
///   1. no grouping keys specified
///   2. rows in ASCENDING order.
/// Treat as one single group.
template <typename T>
std::unique_ptr<column> range_window_ASC(column_view const& input,
                                         column_view const& orderby_column,
                                         T preceding_window,
                                         bool preceding_window_is_unbounded,
                                         T following_window,
                                         bool following_window_is_unbounded,
                                         size_type min_periods,
                                         rolling_aggregation const& aggr,
                                         rmm::cuda_stream_view stream,
                                         rmm::mr::device_memory_resource* mr)
{
  auto [h_nulls_begin_idx, h_nulls_end_idx] = get_null_bounds_for_orderby_column(orderby_column);

  auto preceding_calculator =
    [nulls_begin_idx = h_nulls_begin_idx,
     nulls_end_idx   = h_nulls_end_idx,
     d_orderby       = orderby_column.data<T>(),
     preceding_window,
     preceding_window_is_unbounded] __device__(size_type idx) -> size_type {
    if (preceding_window_is_unbounded) {
      return idx + 1;  // Technically `idx - 0 + 1`,
                       // where 0 == Group start,
                       // and   1 accounts for the current row
    }
    if (idx >= nulls_begin_idx && idx < nulls_end_idx) {
      // Current row is in the null group.
      // Must consider beginning of null-group as window start.
      return idx - nulls_begin_idx + 1;
    }

    // orderby[idx] not null. Binary search the group, excluding null group.
    // If nulls_begin_idx == 0, either
    //  1. NULLS FIRST ordering: Binary search starts where nulls_end_idx.
    //  2. NO NULLS: Binary search starts at 0 (also nulls_end_idx).
    // Otherwise, NULLS LAST ordering. Start at 0.
    auto group_start      = nulls_begin_idx == 0 ? nulls_end_idx : 0;
    auto lowest_in_window = subtract_safe(d_orderby[idx], preceding_window);

    return ((d_orderby + idx) - thrust::lower_bound(thrust::seq,
                                                    d_orderby + group_start,
                                                    d_orderby + idx,
                                                    lowest_in_window)) +
           1;  // Add 1, for `preceding` to account for current row.
  };

  auto preceding_column = expand_to_column(preceding_calculator, input.size(), stream, mr);

  auto following_calculator =
    [nulls_begin_idx = h_nulls_begin_idx,
     nulls_end_idx   = h_nulls_end_idx,
     num_rows        = input.size(),
     d_orderby       = orderby_column.data<T>(),
     following_window,
     following_window_is_unbounded] __device__(size_type idx) -> size_type {
    if (following_window_is_unbounded) { return num_rows - idx - 1; }
    if (idx >= nulls_begin_idx && idx < nulls_end_idx) {
      // Current row is in the null group.
      // Window ends at the end of the null group.
      return nulls_end_idx - idx - 1;
    }

    // orderby[idx] not null. Binary search the group, excluding null group.
    // If nulls_begin_idx == 0, either
    //  1. NULLS FIRST ordering: Binary search ends at num_rows.
    //  2. NO NULLS: Binary search also ends at num_rows.
    // Otherwise, NULLS LAST ordering. End at nulls_begin_idx.

    auto group_end         = nulls_begin_idx == 0 ? num_rows : nulls_begin_idx;
    auto highest_in_window = add_safe(d_orderby[idx], following_window);

    return (thrust::upper_bound(
              thrust::seq, d_orderby + idx, d_orderby + group_end, highest_in_window) -
            (d_orderby + idx)) -
           1;
  };

  auto following_column = expand_to_column(following_calculator, input.size(), stream, mr);

  return cudf::detail::rolling_window(
    input, preceding_column->view(), following_column->view(), min_periods, aggr, stream, mr);
}

// Given an orderby column grouped as specified in group_offsets,
// return the following two vectors:
//  1. Vector with one entry per group, indicating the offset in the group
//     where the null values begin.
//  2. Vector with one entry per group, indicating the offset in the group
//     where the null values end. (i.e. 1 past the last null.)
// Each group in the input orderby column must be sorted,
// with null values clustered at either the start or the end of each group.
// If there are no nulls for any given group, (nulls_begin, nulls_end) == (0,0).
std::tuple<rmm::device_uvector<size_type>, rmm::device_uvector<size_type>>
get_null_bounds_for_orderby_column(column_view const& orderby_column,
                                   cudf::device_span<size_type const> group_offsets,
                                   rmm::cuda_stream_view stream)
{
  // For each group, the null values are clustered at the beginning or the end of the group.
  // These nulls cannot participate, except in their own window.

  auto num_groups = group_offsets.size() - 1;

  if (orderby_column.has_nulls()) {
    auto null_start = rmm::device_uvector<size_type>(num_groups, stream);
    auto null_end   = rmm::device_uvector<size_type>(num_groups, stream);

    auto p_orderby_device_view = column_device_view::create(orderby_column);

    // Null timestamps exist. Find null bounds, per group.
    thrust::for_each(
      rmm::exec_policy(stream),
      thrust::make_counting_iterator(static_cast<size_type>(0)),
      thrust::make_counting_iterator(static_cast<size_type>(num_groups)),
      [d_orderby       = *p_orderby_device_view,
       d_group_offsets = group_offsets.data(),
       d_null_start    = null_start.data(),
       d_null_end      = null_end.data()] __device__(auto group_label) {
        auto group_start           = d_group_offsets[group_label];
        auto group_end             = d_group_offsets[group_label + 1];
        auto first_element_is_null = d_orderby.is_null_nocheck(group_start);
        auto last_element_is_null  = d_orderby.is_null_nocheck(group_end - 1);
        if (!first_element_is_null && !last_element_is_null) {
          // Short circuit: No nulls.
          d_null_start[group_label] = group_start;
          d_null_end[group_label]   = group_start;
        } else if (first_element_is_null && last_element_is_null) {
          // Short circuit: All nulls.
          d_null_start[group_label] = group_start;
          d_null_end[group_label]   = group_end;
        } else if (first_element_is_null) {
          // NULLS FIRST.
          d_null_start[group_label] = group_start;
          d_null_end[group_label]   = *thrust::partition_point(
            thrust::seq,
            thrust::make_counting_iterator(group_start),
            thrust::make_counting_iterator(group_end),
            [&d_orderby] __device__(auto i) { return d_orderby.is_null_nocheck(i); });
        } else {
          // NULLS LAST.
          d_null_end[group_label]   = group_end;
          d_null_start[group_label] = *thrust::partition_point(
            thrust::seq,
            thrust::make_counting_iterator(group_start),
            thrust::make_counting_iterator(group_end),
            [&d_orderby] __device__(auto i) { return d_orderby.is_valid_nocheck(i); });
        }
      });

    return std::make_tuple(std::move(null_start), std::move(null_end));
  } else {
    // The returned vectors have num_groups items, but the input offsets have num_groups+1
    // Drop the last element using a span
    auto group_offsets_span =
      cudf::device_span<cudf::size_type const>(group_offsets.data(), num_groups);

    // When there are no nulls, just copy the input group offsets to the output.
    return std::make_tuple(cudf::detail::make_device_uvector_async(group_offsets_span, stream),
                           cudf::detail::make_device_uvector_async(group_offsets_span, stream));
  }
}

// Range window computation, for orderby column in ASCENDING order.
template <typename T>
std::unique_ptr<column> range_window_ASC(column_view const& input,
                                         column_view const& orderby_column,
                                         rmm::device_uvector<cudf::size_type> const& group_offsets,
                                         rmm::device_uvector<cudf::size_type> const& group_labels,
                                         T preceding_window,
                                         bool preceding_window_is_unbounded,
                                         T following_window,
                                         bool following_window_is_unbounded,
                                         size_type min_periods,
                                         rolling_aggregation const& aggr,
                                         rmm::cuda_stream_view stream,
                                         rmm::mr::device_memory_resource* mr)
{
  auto [null_start, null_end] =
    get_null_bounds_for_orderby_column(orderby_column, group_offsets, stream);

  auto preceding_calculator =
    [d_group_offsets = group_offsets.data(),
     d_group_labels  = group_labels.data(),
     d_orderby       = orderby_column.data<T>(),
     d_nulls_begin   = null_start.data(),
     d_nulls_end     = null_end.data(),
     preceding_window,
     preceding_window_is_unbounded] __device__(size_type idx) -> size_type {
    auto group_label = d_group_labels[idx];
    auto group_start = d_group_offsets[group_label];
    auto nulls_begin = d_nulls_begin[group_label];
    auto nulls_end   = d_nulls_end[group_label];

    if (preceding_window_is_unbounded) { return idx - group_start + 1; }

    // If idx lies in the null-range, the window is the null range.
    if (idx >= nulls_begin && idx < nulls_end) {
      // Current row is in the null group.
      // The window starts at the start of the null group.
      return idx - nulls_begin + 1;
    }

    // orderby[idx] not null. Search must exclude the null group.
    // If nulls_begin == group_start, either of the following is true:
    //  1. NULLS FIRST ordering: Search must begin at nulls_end.
    //  2. NO NULLS: Search must begin at group_start (which also equals nulls_end.)
    // Otherwise, NULLS LAST ordering. Search must start at nulls group_start.
    auto search_start = nulls_begin == group_start ? nulls_end : group_start;

    auto lowest_in_window = subtract_safe(d_orderby[idx], preceding_window);

    return ((d_orderby + idx) - thrust::lower_bound(thrust::seq,
                                                    d_orderby + search_start,
                                                    d_orderby + idx,
                                                    lowest_in_window)) +
           1;  // Add 1, for `preceding` to account for current row.
  };

  auto preceding_column = expand_to_column(preceding_calculator, input.size(), stream, mr);

  auto following_calculator =
    [d_group_offsets = group_offsets.data(),
     d_group_labels  = group_labels.data(),
     d_orderby       = orderby_column.data<T>(),
     d_nulls_begin   = null_start.data(),
     d_nulls_end     = null_end.data(),
     following_window,
     following_window_is_unbounded] __device__(size_type idx) -> size_type {
    auto group_label = d_group_labels[idx];
    auto group_start = d_group_offsets[group_label];
    auto group_end   = d_group_offsets[group_label + 1];  // Cannot fall off the end, since offsets
                                                          // is capped with `input.size()`.
    auto nulls_begin = d_nulls_begin[group_label];
    auto nulls_end   = d_nulls_end[group_label];

    if (following_window_is_unbounded) { return (group_end - idx) - 1; }

    // If idx lies in the null-range, the window is the null range.
    if (idx >= nulls_begin && idx < nulls_end) {
      // Current row is in the null group.
      // The window ends at the end of the null group.
      return nulls_end - idx - 1;
    }

    // orderby[idx] not null. Search must exclude the null group.
    // If nulls_begin == group_start, either of the following is true:
    //  1. NULLS FIRST ordering: Search ends at group_end.
    //  2. NO NULLS: Search ends at group_end.
    // Otherwise, NULLS LAST ordering. Search ends at nulls_begin.
    auto search_end = nulls_begin == group_start ? group_end : nulls_begin;

    auto highest_in_window = add_safe(d_orderby[idx], following_window);

    return (thrust::upper_bound(
              thrust::seq, d_orderby + idx, d_orderby + search_end, highest_in_window) -
            (d_orderby + idx)) -
           1;
  };

  auto following_column = expand_to_column(following_calculator, input.size(), stream, mr);

  return cudf::detail::rolling_window(
    input, preceding_column->view(), following_column->view(), min_periods, aggr, stream, mr);
}

/// Range window computation, with
///   1. no grouping keys specified
///   2. rows in DESCENDING order.
/// Treat as one single group.
template <typename T>
std::unique_ptr<column> range_window_DESC(column_view const& input,
                                          column_view const& orderby_column,
                                          T preceding_window,
                                          bool preceding_window_is_unbounded,
                                          T following_window,
                                          bool following_window_is_unbounded,
                                          size_type min_periods,
                                          rolling_aggregation const& aggr,
                                          rmm::cuda_stream_view stream,
                                          rmm::mr::device_memory_resource* mr)
{
  auto [h_nulls_begin_idx, h_nulls_end_idx] = get_null_bounds_for_orderby_column(orderby_column);

  auto preceding_calculator =
    [nulls_begin_idx = h_nulls_begin_idx,
     nulls_end_idx   = h_nulls_end_idx,
     d_orderby       = orderby_column.data<T>(),
     preceding_window,
     preceding_window_is_unbounded] __device__(size_type idx) -> size_type {
    if (preceding_window_is_unbounded) {
      return idx + 1;  // Technically `idx - 0 + 1`,
                       // where 0 == Group start,
                       // and   1 accounts for the current row
    }
    if (idx >= nulls_begin_idx && idx < nulls_end_idx) {
      // Current row is in the null group.
      // Must consider beginning of null-group as window start.
      return idx - nulls_begin_idx + 1;
    }

    // orderby[idx] not null. Binary search the group, excluding null group.
    // If nulls_begin_idx == 0, either
    //  1. NULLS FIRST ordering: Binary search starts where nulls_end_idx.
    //  2. NO NULLS: Binary search starts at 0 (also nulls_end_idx).
    // Otherwise, NULLS LAST ordering. Start at 0.
    auto group_start       = nulls_begin_idx == 0 ? nulls_end_idx : 0;
    auto highest_in_window = add_safe(d_orderby[idx], preceding_window);

    return ((d_orderby + idx) -
            thrust::lower_bound(thrust::seq,
                                d_orderby + group_start,
                                d_orderby + idx,
                                highest_in_window,
                                thrust::greater<decltype(highest_in_window)>())) +
           1;  // Add 1, for `preceding` to account for current row.
  };

  auto preceding_column = expand_to_column(preceding_calculator, input.size(), stream, mr);

  auto following_calculator =
    [nulls_begin_idx = h_nulls_begin_idx,
     nulls_end_idx   = h_nulls_end_idx,
     num_rows        = input.size(),
     d_orderby       = orderby_column.data<T>(),
     following_window,
     following_window_is_unbounded] __device__(size_type idx) -> size_type {
    if (following_window_is_unbounded) { return (num_rows - idx) - 1; }
    if (idx >= nulls_begin_idx && idx < nulls_end_idx) {
      // Current row is in the null group.
      // Window ends at the end of the null group.
      return nulls_end_idx - idx - 1;
    }

    // orderby[idx] not null. Search must exclude null group.
    // If nulls_begin_idx = 0, either
    //  1. NULLS FIRST ordering: Search ends at num_rows.
    //  2. NO NULLS: Search also ends at num_rows.
    // Otherwise, NULLS LAST ordering: End at nulls_begin_idx.

    auto group_end        = nulls_begin_idx == 0 ? num_rows : nulls_begin_idx;
    auto lowest_in_window = subtract_safe(d_orderby[idx], following_window);

    return (thrust::upper_bound(thrust::seq,
                                d_orderby + idx,
                                d_orderby + group_end,
                                lowest_in_window,
                                thrust::greater<decltype(lowest_in_window)>()) -
            (d_orderby + idx)) -
           1;
  };

  auto following_column = expand_to_column(following_calculator, input.size(), stream, mr);

  return cudf::detail::rolling_window(
    input, preceding_column->view(), following_column->view(), min_periods, aggr, stream, mr);
}

// Range window computation, for rows in DESCENDING order.
template <typename T>
std::unique_ptr<column> range_window_DESC(column_view const& input,
                                          column_view const& orderby_column,
                                          rmm::device_uvector<cudf::size_type> const& group_offsets,
                                          rmm::device_uvector<cudf::size_type> const& group_labels,
                                          T preceding_window,
                                          bool preceding_window_is_unbounded,
                                          T following_window,
                                          bool following_window_is_unbounded,
                                          size_type min_periods,
                                          rolling_aggregation const& aggr,
                                          rmm::cuda_stream_view stream,
                                          rmm::mr::device_memory_resource* mr)
{
  auto [null_start, null_end] =
    get_null_bounds_for_orderby_column(orderby_column, group_offsets, stream);

  auto preceding_calculator =
    [d_group_offsets = group_offsets.data(),
     d_group_labels  = group_labels.data(),
     d_orderby       = orderby_column.data<T>(),
     d_nulls_begin   = null_start.data(),
     d_nulls_end     = null_end.data(),
     preceding_window,
     preceding_window_is_unbounded] __device__(size_type idx) -> size_type {
    auto group_label = d_group_labels[idx];
    auto group_start = d_group_offsets[group_label];
    auto nulls_begin = d_nulls_begin[group_label];
    auto nulls_end   = d_nulls_end[group_label];

    if (preceding_window_is_unbounded) { return (idx - group_start) + 1; }

    // If idx lies in the null-range, the window is the null range.
    if (idx >= nulls_begin && idx < nulls_end) {
      // Current row is in the null group.
      // The window starts at the start of the null group.
      return idx - nulls_begin + 1;
    }

    // orderby[idx] not null. Search must exclude the null group.
    // If nulls_begin == group_start, either of the following is true:
    //  1. NULLS FIRST ordering: Search must begin at nulls_end.
    //  2. NO NULLS: Search must begin at group_start (which also equals nulls_end.)
    // Otherwise, NULLS LAST ordering. Search must start at nulls group_start.
    auto search_start = nulls_begin == group_start ? nulls_end : group_start;

    auto highest_in_window = add_safe(d_orderby[idx], preceding_window);

    return ((d_orderby + idx) -
            thrust::lower_bound(thrust::seq,
                                d_orderby + search_start,
                                d_orderby + idx,
                                highest_in_window,
                                thrust::greater<decltype(highest_in_window)>())) +
           1;  // Add 1, for `preceding` to account for current row.
  };

  auto preceding_column = expand_to_column(preceding_calculator, input.size(), stream, mr);

  auto following_calculator =
    [d_group_offsets = group_offsets.data(),
     d_group_labels  = group_labels.data(),
     d_orderby       = orderby_column.data<T>(),
     d_nulls_begin   = null_start.data(),
     d_nulls_end     = null_end.data(),
     following_window,
     following_window_is_unbounded] __device__(size_type idx) -> size_type {
    auto group_label = d_group_labels[idx];
    auto group_start = d_group_offsets[group_label];
    auto group_end   = d_group_offsets[group_label + 1];
    auto nulls_begin = d_nulls_begin[group_label];
    auto nulls_end   = d_nulls_end[group_label];

    if (following_window_is_unbounded) { return (group_end - idx) - 1; }

    // If idx lies in the null-range, the window is the null range.
    if (idx >= nulls_begin && idx < nulls_end) {
      // Current row is in the null group.
      // The window ends at the end of the null group.
      return nulls_end - idx - 1;
    }

    // orderby[idx] not null. Search must exclude the null group.
    // If nulls_begin == group_start, either of the following is true:
    //  1. NULLS FIRST ordering: Search ends at group_end.
    //  2. NO NULLS: Search ends at group_end.
    // Otherwise, NULLS LAST ordering. Search ends at nulls_begin.
    auto search_end = nulls_begin == group_start ? group_end : nulls_begin;

    auto lowest_in_window = subtract_safe(d_orderby[idx], following_window);

    return (thrust::upper_bound(thrust::seq,
                                d_orderby + idx,
                                d_orderby + search_end,
                                lowest_in_window,
                                thrust::greater<decltype(lowest_in_window)>()) -
            (d_orderby + idx)) -
           1;
  };

  auto following_column = expand_to_column(following_calculator, input.size(), stream, mr);

  if (aggr.kind == aggregation::CUDA || aggr.kind == aggregation::PTX) {
    CUDF_FAIL("Ranged rolling window does NOT (yet) support UDF.");
  } else {
    return cudf::detail::rolling_window(
      input, preceding_column->view(), following_column->view(), min_periods, aggr, stream, mr);
  }
}

template <typename OrderByT>
std::unique_ptr<column> grouped_range_rolling_window_impl(
  column_view const& input,
  column_view const& orderby_column,
  cudf::order const& timestamp_ordering,
  rmm::device_uvector<cudf::size_type> const& group_offsets,
  rmm::device_uvector<cudf::size_type> const& group_labels,
  range_window_bounds const& preceding_window,
  range_window_bounds const& following_window,
  size_type min_periods,
  rolling_aggregation const& aggr,
  rmm::cuda_stream_view stream,
  rmm::mr::device_memory_resource* mr)
{
  auto preceding_value = detail::range_comparable_value<OrderByT>(preceding_window);
  auto following_value = detail::range_comparable_value<OrderByT>(following_window);

  if (timestamp_ordering == cudf::order::ASCENDING) {
    return group_offsets.is_empty() ? range_window_ASC(input,
                                                       orderby_column,
                                                       preceding_value,
                                                       preceding_window.is_unbounded(),
                                                       following_value,
                                                       following_window.is_unbounded(),
                                                       min_periods,
                                                       aggr,
                                                       stream,
                                                       mr)
                                    : range_window_ASC(input,
                                                       orderby_column,
                                                       group_offsets,
                                                       group_labels,
                                                       preceding_value,
                                                       preceding_window.is_unbounded(),
                                                       following_value,
                                                       following_window.is_unbounded(),
                                                       min_periods,
                                                       aggr,
                                                       stream,
                                                       mr);
  } else {
    return group_offsets.is_empty() ? range_window_DESC(input,
                                                        orderby_column,
                                                        preceding_value,
                                                        preceding_window.is_unbounded(),
                                                        following_value,
                                                        following_window.is_unbounded(),
                                                        min_periods,
                                                        aggr,
                                                        stream,
                                                        mr)
                                    : range_window_DESC(input,
                                                        orderby_column,
                                                        group_offsets,
                                                        group_labels,
                                                        preceding_value,
                                                        preceding_window.is_unbounded(),
                                                        following_value,
                                                        following_window.is_unbounded(),
                                                        min_periods,
                                                        aggr,
                                                        stream,
                                                        mr);
  }
}

struct dispatch_grouped_range_rolling_window {
  template <typename OrderByColumnType, typename... Args>
  std::enable_if_t<!detail::is_supported_order_by_column_type<OrderByColumnType>(),
                   std::unique_ptr<column>>
  operator()(Args&&...) const
  {
    CUDF_FAIL("Unsupported OrderBy column type.");
  }

  template <typename OrderByColumnType>
  std::enable_if_t<detail::is_supported_order_by_column_type<OrderByColumnType>(),
                   std::unique_ptr<column>>
  operator()(column_view const& input,
             column_view const& orderby_column,
             cudf::order const& timestamp_ordering,
             rmm::device_uvector<cudf::size_type> const& group_offsets,
             rmm::device_uvector<cudf::size_type> const& group_labels,
             range_window_bounds const& preceding_window,
             range_window_bounds const& following_window,
             size_type min_periods,
             rolling_aggregation const& aggr,
             rmm::cuda_stream_view stream,
             rmm::mr::device_memory_resource* mr) const
  {
    return grouped_range_rolling_window_impl<OrderByColumnType>(input,
                                                                orderby_column,
                                                                timestamp_ordering,
                                                                group_offsets,
                                                                group_labels,
                                                                preceding_window,
                                                                following_window,
                                                                min_periods,
                                                                aggr,
                                                                stream,
                                                                mr);
  }
};

/**
 * @brief Functor to convert from size_type (number of days) to appropriate duration type.
 */
struct to_duration_bounds {
  template <typename OrderBy, std::enable_if_t<cudf::is_timestamp<OrderBy>(), void>* = nullptr>
  range_window_bounds operator()(size_type num_days) const
  {
    using DurationT = typename OrderBy::duration;
    return range_window_bounds::get(duration_scalar<DurationT>{duration_D{num_days}, true});
  }

  template <typename OrderBy, std::enable_if_t<!cudf::is_timestamp<OrderBy>(), void>* = nullptr>
  range_window_bounds operator()(size_type) const
  {
    CUDF_FAIL("Expected timestamp orderby column.");
  }
};

/**
 * @brief Get duration type corresponding to specified timestamp type.
 */
data_type get_duration_type_for(cudf::data_type timestamp_type)
{
  switch (timestamp_type.id()) {
    case type_id::TIMESTAMP_DAYS: return data_type{type_id::DURATION_DAYS};
    case type_id::TIMESTAMP_SECONDS: return data_type{type_id::DURATION_SECONDS};
    case type_id::TIMESTAMP_MILLISECONDS: return data_type{type_id::DURATION_MILLISECONDS};
    case type_id::TIMESTAMP_MICROSECONDS: return data_type{type_id::DURATION_MICROSECONDS};
    case type_id::TIMESTAMP_NANOSECONDS: return data_type{type_id::DURATION_NANOSECONDS};
    default: CUDF_FAIL("Expected timestamp orderby column.");
  }
}

/**
 * @brief Bridge function to convert from size_type (number of days) to appropriate duration type.
 *
 * This helps adapt the old `grouped_time_range_rolling_window()` functions that took a "number of
 * days" to the new `range_window_bounds` interface.
 *
 * @param num_days Window bounds specified in number of days in `size_type`
 * @param timestamp_type Data-type of the orderby column to which the `num_days` is to be adapted.
 * @return range_window_bounds A `range_window_bounds` to be used with the new API.
 */
range_window_bounds to_range_bounds(cudf::size_type num_days, cudf::data_type timestamp_type)
{
  return cudf::type_dispatcher(timestamp_type, to_duration_bounds{}, num_days);
}

/**
 * @brief Bridge function to convert from `window_bounds` (in days) to appropriate duration type.
 *
 * This helps adapt the old `grouped_time_range_rolling_window()` functions that took a
 * `window_bounds` to the new `range_window_bounds` interface.
 *
 * @param days_bounds The static window-width `window_bounds` object
 * @param timestamp_type Data-type of the orderby column to which the `num_days` is to be adapted.
 * @return range_window_bounds A `range_window_bounds` to be used with the new API.
 */
range_window_bounds to_range_bounds(cudf::window_bounds const& days_bounds,
                                    cudf::data_type timestamp_type)
{
  return days_bounds.is_unbounded
           ? range_window_bounds::unbounded(get_duration_type_for(timestamp_type))
           : cudf::type_dispatcher(timestamp_type, to_duration_bounds{}, days_bounds.value);
}

}  // namespace

namespace detail {

/**
 * @copydoc  std::unique_ptr<column> grouped_range_rolling_window(
 *               table_view const& group_keys,
 *               column_view const& orderby_column,
 *               cudf::order const& order,
 *               column_view const& input,
 *               range_window_bounds const& preceding,
 *               range_window_bounds const& following,
 *               size_type min_periods,
 *               rolling_aggregation const& aggr,
 *               rmm::mr::device_memory_resource* mr );
 *
 * @param stream CUDA stream used for device memory operations and kernel launches.
 */
std::unique_ptr<column> grouped_range_rolling_window(table_view const& group_keys,
                                                     column_view const& order_by_column,
                                                     cudf::order const& order,
                                                     column_view const& input,
                                                     range_window_bounds const& preceding,
                                                     range_window_bounds const& following,
                                                     size_type min_periods,
                                                     rolling_aggregation const& aggr,
                                                     rmm::cuda_stream_view stream,
                                                     rmm::mr::device_memory_resource* mr)
{
  CUDF_FUNC_RANGE();

  if (input.is_empty()) { return cudf::detail::empty_output_for_rolling_aggregation(input, aggr); }

  CUDF_EXPECTS((group_keys.num_columns() == 0 || group_keys.num_rows() == input.size()),
               "Size mismatch between group_keys and input vector.");

  CUDF_EXPECTS((min_periods > 0), "min_periods must be positive");

  using sort_groupby_helper = cudf::groupby::detail::sort::sort_groupby_helper;
  using index_vector        = sort_groupby_helper::index_vector;

  index_vector group_offsets(0, stream), group_labels(0, stream);
  if (group_keys.num_columns() > 0) {
    sort_groupby_helper helper{group_keys, cudf::null_policy::INCLUDE, cudf::sorted::YES};
    group_offsets = index_vector(helper.group_offsets(stream), stream);
    group_labels  = index_vector(helper.group_labels(stream), stream);
  }

  return cudf::type_dispatcher(order_by_column.type(),
                               dispatch_grouped_range_rolling_window{},
                               input,
                               order_by_column,
                               order,
                               group_offsets,
                               group_labels,
                               preceding,
                               following,
                               min_periods,
                               aggr,
                               stream,
                               mr);
}

}  // namespace detail

/**
 * @copydoc std::unique_ptr<column> grouped_time_range_rolling_window(
 *              table_view const& group_keys,
 *              column_view const& timestamp_column,
 *              cudf::order const& timestamp_order,
 *              column_view const& input,
 *              size_type preceding_window_in_days,
 *              size_type following_window_in_days,
 *              size_type min_periods,
 *              rolling_aggregation const& aggr,
 *              rmm::mr::device_memory_resource* mr);
 */
std::unique_ptr<column> grouped_time_range_rolling_window(table_view const& group_keys,
                                                          column_view const& timestamp_column,
                                                          cudf::order const& timestamp_order,
                                                          column_view const& input,
                                                          size_type preceding_window_in_days,
                                                          size_type following_window_in_days,
                                                          size_type min_periods,
                                                          rolling_aggregation const& aggr,
                                                          rmm::mr::device_memory_resource* mr)
{
  auto preceding = to_range_bounds(preceding_window_in_days, timestamp_column.type());
  auto following = to_range_bounds(following_window_in_days, timestamp_column.type());

  return grouped_range_rolling_window(group_keys,
                                      timestamp_column,
                                      timestamp_order,
                                      input,
                                      preceding,
                                      following,
                                      min_periods,
                                      aggr,
                                      mr);
}

/**
 * @copydoc std::unique_ptr<column> grouped_time_range_rolling_window(
 *            table_view const& group_keys,
 *            column_view const& timestamp_column,
 *            cudf::order const& timestamp_order,
 *            column_view const& input,
 *            window_bounds preceding_window_in_days,
 *            window_bounds following_window_in_days,
 *            size_type min_periods,
 *            rolling_aggregation const& aggr,
 *            rmm::mr::device_memory_resource* mr);
 */
std::unique_ptr<column> grouped_time_range_rolling_window(table_view const& group_keys,
                                                          column_view const& timestamp_column,
                                                          cudf::order const& timestamp_order,
                                                          column_view const& input,
                                                          window_bounds preceding_window_in_days,
                                                          window_bounds following_window_in_days,
                                                          size_type min_periods,
                                                          rolling_aggregation const& aggr,
                                                          rmm::mr::device_memory_resource* mr)
{
  range_window_bounds preceding =
    to_range_bounds(preceding_window_in_days, timestamp_column.type());
  range_window_bounds following =
    to_range_bounds(following_window_in_days, timestamp_column.type());

  return grouped_range_rolling_window(group_keys,
                                      timestamp_column,
                                      timestamp_order,
                                      input,
                                      preceding,
                                      following,
                                      min_periods,
                                      aggr,
                                      rmm::cuda_stream_default,
                                      mr);
}

/**
 * @copydoc  std::unique_ptr<column> grouped_range_rolling_window(
 *               table_view const& group_keys,
 *               column_view const& orderby_column,
 *               cudf::order const& order,
 *               column_view const& input,
 *               range_window_bounds const& preceding,
 *               range_window_bounds const& following,
 *               size_type min_periods,
 *               rolling_aggregation const& aggr,
 *               rmm::mr::device_memory_resource* mr );
 */
std::unique_ptr<column> grouped_range_rolling_window(table_view const& group_keys,
                                                     column_view const& timestamp_column,
                                                     cudf::order const& timestamp_order,
                                                     column_view const& input,
                                                     range_window_bounds const& preceding,
                                                     range_window_bounds const& following,
                                                     size_type min_periods,
                                                     rolling_aggregation const& aggr,
                                                     rmm::mr::device_memory_resource* mr)
{
  return detail::grouped_range_rolling_window(group_keys,
                                              timestamp_column,
                                              timestamp_order,
                                              input,
                                              preceding,
                                              following,
                                              min_periods,
                                              aggr,
                                              rmm::cuda_stream_default,
                                              mr);
}

}  // namespace cudf
