/*
 * Copyright (c) 2020, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <cudf/unary.hpp>
#include "rolling_detail.cuh"

namespace cudf {

std::unique_ptr<column> grouped_rolling_window(table_view const& group_keys,
                                               column_view const& input,
                                               size_type preceding_window,
                                               size_type following_window,
                                               size_type min_periods,
                                               std::unique_ptr<aggregation> const& aggr,
                                               rmm::mr::device_memory_resource* mr)
{
  return grouped_rolling_window(group_keys,
                                input,
                                window_bounds::get(preceding_window),
                                window_bounds::get(following_window),
                                min_periods,
                                aggr,
                                mr);
}

std::unique_ptr<column> grouped_rolling_window(table_view const& group_keys,
                                               column_view const& input,
                                               window_bounds preceding_window,
                                               window_bounds following_window,
                                               size_type min_periods,
                                               std::unique_ptr<aggregation> const& aggr,
                                               rmm::mr::device_memory_resource* mr)
{
  return grouped_rolling_window(group_keys,
                                input,
                                empty_like(input)->view(),
                                preceding_window,
                                following_window,
                                min_periods,
                                aggr,
                                mr);
}

std::unique_ptr<column> grouped_rolling_window(table_view const& group_keys,
                                               column_view const& input,
                                               column_view const& default_outputs,
                                               size_type preceding_window,
                                               size_type following_window,
                                               size_type min_periods,
                                               std::unique_ptr<aggregation> const& aggr,
                                               rmm::mr::device_memory_resource* mr)
{
  return grouped_rolling_window(group_keys,
                                input,
                                default_outputs,
                                window_bounds::get(preceding_window),
                                window_bounds::get(following_window),
                                min_periods,
                                aggr,
                                mr);
}

namespace detail {

std::unique_ptr<column> grouped_rolling_window(table_view const& group_keys,
                                               column_view const& input,
                                               column_view const& default_outputs,
                                               window_bounds preceding_window_bounds,
                                               window_bounds following_window_bounds,
                                               size_type min_periods,
                                               std::unique_ptr<aggregation> const& aggr,
                                               rmm::cuda_stream_view stream,
                                               rmm::mr::device_memory_resource* mr)
{
  CUDF_FUNC_RANGE();

  if (input.is_empty()) return empty_like(input);

  CUDF_EXPECTS((group_keys.num_columns() == 0 || group_keys.num_rows() == input.size()),
               "Size mismatch between group_keys and input vector.");

  CUDF_EXPECTS((min_periods > 0), "min_periods must be positive");

  CUDF_EXPECTS((default_outputs.is_empty() || default_outputs.size() == input.size()),
               "Defaults column must be either empty or have as many rows as the input column.");

  auto const preceding_window = preceding_window_bounds.value;
  auto const following_window = following_window_bounds.value;

  if (group_keys.num_columns() == 0) {
    // No Groupby columns specified. Treat as one big group.
    return rolling_window(
      input, default_outputs, preceding_window, following_window, min_periods, aggr, mr);
  }

  using sort_groupby_helper = cudf::groupby::detail::sort::sort_groupby_helper;

  sort_groupby_helper helper{group_keys, cudf::null_policy::INCLUDE, cudf::sorted::YES};
  auto group_offsets{helper.group_offsets()};
  auto const& group_labels{helper.group_labels()};

  // `group_offsets` are interpreted in adjacent pairs, each pair representing the offsets
  // of the first, and one past the last elements in a group.
  //
  // If `group_offsets` is not empty, it must contain at least two offsets:
  //   a. 0, indicating the first element in `input`
  //   b. input.size(), indicating one past the last element in `input`.
  //
  // Thus, for an input of 1000 rows,
  //   0. [] indicates a single group, spanning the entire column.
  //   1  [10] is invalid.
  //   2. [0, 1000] indicates a single group, spanning the entire column (thus, equivalent to no
  //   groups.)
  //   3. [0, 500, 1000] indicates two equal-sized groups: [0,500), and [500,1000).

  assert(group_offsets.size() >= 2 && group_offsets[0] == 0 &&
         group_offsets[group_offsets.size() - 1] == input.size() &&
         "Must have at least one group.");

  auto preceding_calculator = [d_group_offsets = group_offsets.data().get(),
                               d_group_labels  = group_labels.data().get(),
                               preceding_window] __device__(size_type idx) {
    auto group_label = d_group_labels[idx];
    auto group_start = d_group_offsets[group_label];
    return thrust::minimum<size_type>{}(preceding_window,
                                        idx - group_start + 1);  // Preceding includes current row.
  };

  auto following_calculator = [d_group_offsets = group_offsets.data().get(),
                               d_group_labels  = group_labels.data().get(),
                               following_window] __device__(size_type idx) {
    auto group_label = d_group_labels[idx];
    auto group_end =
      d_group_offsets[group_label +
                      1];  // Cannot fall off the end, since offsets is capped with `input.size()`.
    return thrust::minimum<size_type>{}(following_window, (group_end - 1) - idx);
  };

  if (aggr->kind == aggregation::CUDA || aggr->kind == aggregation::PTX) {
    cudf::detail::preceding_window_wrapper grouped_preceding_window{
      group_offsets.data().get(), group_labels.data().get(), preceding_window};

    cudf::detail::following_window_wrapper grouped_following_window{
      group_offsets.data().get(), group_labels.data().get(), following_window};

    return cudf::detail::rolling_window_udf(input,
                                            grouped_preceding_window,
                                            "cudf::detail::preceding_window_wrapper",
                                            grouped_following_window,
                                            "cudf::detail::following_window_wrapper",
                                            min_periods,
                                            aggr,
                                            stream,
                                            mr);
  } else {
    return cudf::detail::rolling_window(
      input,
      default_outputs,
      thrust::make_transform_iterator(thrust::make_counting_iterator<size_type>(0),
                                      preceding_calculator),
      thrust::make_transform_iterator(thrust::make_counting_iterator<size_type>(0),
                                      following_calculator),
      min_periods,
      aggr,
      stream,
      mr);
  }
}

}  // namespace detail

std::unique_ptr<column> grouped_rolling_window(table_view const& group_keys,
                                               column_view const& input,
                                               column_view const& default_outputs,
                                               window_bounds preceding_window_bounds,
                                               window_bounds following_window_bounds,
                                               size_type min_periods,
                                               std::unique_ptr<aggregation> const& aggr,
                                               rmm::mr::device_memory_resource* mr)
{
  return detail::grouped_rolling_window(group_keys,
                                        input,
                                        default_outputs,
                                        preceding_window_bounds,
                                        following_window_bounds,
                                        min_periods,
                                        aggr,
                                        rmm::cuda_stream_default,
                                        mr);
}

namespace {

bool is_supported_range_frame_unit(cudf::data_type const& data_type)
{
  auto id = data_type.id();
  return id == cudf::type_id::TIMESTAMP_DAYS || id == cudf::type_id::TIMESTAMP_SECONDS ||
         id == cudf::type_id::TIMESTAMP_MILLISECONDS ||
         id == cudf::type_id::TIMESTAMP_MICROSECONDS || id == cudf::type_id::TIMESTAMP_NANOSECONDS;
}

/// Fetches multiplication factor to normalize window sizes, depending on the datatype of the
/// timestamp column. Used for time-based rolling-window operations. E.g. If the timestamp column is
/// in TIMESTAMP_SECONDS, and the window sizes are specified in DAYS, the window size needs to be
/// multiplied by `24*60*60`, before comparisons with the timestamps.
size_t multiplication_factor(cudf::data_type const& data_type)
{
  // Assume timestamps.
  switch (data_type.id()) {
    case cudf::type_id::TIMESTAMP_DAYS: return 1L;
    case cudf::type_id::TIMESTAMP_SECONDS: return 24L * 60 * 60;
    case cudf::type_id::TIMESTAMP_MILLISECONDS: return 24L * 60 * 60 * 1000;
    case cudf::type_id::TIMESTAMP_MICROSECONDS: return 24L * 60 * 60 * 1000 * 1000;
    case cudf::type_id::TIMESTAMP_NANOSECONDS: return 24L * 60 * 60 * 1000 * 1000 * 1000;
    default:
      CUDF_FAIL("Unexpected data-type for timestamp-based rolling window operation!");
      return {};
  }
}

/// Given a single, ungrouped timestamp column, return the indices corresponding
/// to the first null timestamp, and (one past) the last null timestamp.
/// The input column is sorted, with all null values clustered either
/// at the beginning of the column or at the end.
/// If no null values are founds, null_begin and null_end are 0.
std::tuple<size_type, size_type> get_null_bounds_for_timestamp_column(
  column_view const& timestamp_column)
{
  auto const num_rows  = timestamp_column.size();
  auto const num_nulls = timestamp_column.null_count();

  if (num_nulls == num_rows || num_nulls == 0) {
    // Short-circuit: All nulls, or no nulls.
    return std::make_tuple(0, num_nulls);
  }

  auto const first_row_is_null = timestamp_column.null_count(0, 1) == 1;

  return first_row_is_null ? std::make_tuple(0, num_nulls)
                           : std::make_tuple(num_rows - num_nulls, num_rows);
}

using TimeT = int64_t;  // Timestamp representations normalized to int64_t.

template <typename Calculator>
std::unique_ptr<column> expand_to_column(Calculator const& calc,
                                         size_type const& num_rows,
                                         rmm::cuda_stream_view stream,
                                         rmm::mr::device_memory_resource* mr)
{
  auto window_column = cudf::make_fixed_width_column(
    cudf::data_type{type_id::INT32}, num_rows, cudf::mask_state::UNALLOCATED, stream, mr);

  auto begin = thrust::make_transform_iterator(thrust::make_counting_iterator<size_type>(0), calc);

  thrust::copy_n(
    rmm::exec_policy(stream), begin, num_rows, window_column->mutable_view().data<size_type>());

  return window_column;
}

/// Time-range window computation, with
///   1. no grouping keys specified
///   2. timetamps in ASCENDING order.
/// Treat as one single group.
std::unique_ptr<column> time_range_window_ASC(column_view const& input,
                                              column_view const& timestamp_column,
                                              TimeT preceding_window,
                                              bool preceding_window_is_unbounded,
                                              TimeT following_window,
                                              bool following_window_is_unbounded,
                                              size_type min_periods,
                                              std::unique_ptr<aggregation> const& aggr,
                                              rmm::cuda_stream_view stream,
                                              rmm::mr::device_memory_resource* mr)
{
  size_type nulls_begin_idx, nulls_end_idx;
  std::tie(nulls_begin_idx, nulls_end_idx) = get_null_bounds_for_timestamp_column(timestamp_column);

  auto preceding_calculator =
    [nulls_begin_idx,
     nulls_end_idx,
     d_timestamps = timestamp_column.data<TimeT>(),
     preceding_window,
     preceding_window_is_unbounded] __device__(size_type idx) -> size_type {
    if (preceding_window_is_unbounded) {
      return idx + 1;  // Technically `idx - 0 + 1`,
                       // where 0 == Group start,
                       // and   1 accounts for the current row
    }
    if (idx >= nulls_begin_idx && idx < nulls_end_idx) {
      // Current row is in the null group.
      // Must consider beginning of null-group as window start.
      return idx - nulls_begin_idx + 1;
    }

    // timestamp[idx] not null. Binary search the group, excluding null group.
    // If nulls_begin_idx == 0, either
    //  1. NULLS FIRST ordering: Binary search starts where nulls_end_idx.
    //  2. NO NULLS: Binary search starts at 0 (also nulls_end_idx).
    // Otherwise, NULLS LAST ordering. Start at 0.
    auto group_start                = nulls_begin_idx == 0 ? nulls_end_idx : 0;
    auto lowest_timestamp_in_window = d_timestamps[idx] - preceding_window;

    return ((d_timestamps + idx) - thrust::lower_bound(thrust::seq,
                                                       d_timestamps + group_start,
                                                       d_timestamps + idx,
                                                       lowest_timestamp_in_window)) +
           1;  // Add 1, for `preceding` to account for current row.
  };

  auto preceding_column = expand_to_column(preceding_calculator, input.size(), stream, mr);

  auto following_calculator =
    [nulls_begin_idx,
     nulls_end_idx,
     num_rows     = input.size(),
     d_timestamps = timestamp_column.data<TimeT>(),
     following_window,
     following_window_is_unbounded] __device__(size_type idx) -> size_type {
    if (following_window_is_unbounded) { return num_rows - idx - 1; }
    if (idx >= nulls_begin_idx && idx < nulls_end_idx) {
      // Current row is in the null group.
      // Window ends at the end of the null group.
      return nulls_end_idx - idx - 1;
    }

    // timestamp[idx] not null. Binary search the group, excluding null group.
    // If nulls_begin_idx == 0, either
    //  1. NULLS FIRST ordering: Binary search ends at num_rows.
    //  2. NO NULLS: Binary search also ends at num_rows.
    // Otherwise, NULLS LAST ordering. End at nulls_begin_idx.

    auto group_end                   = nulls_begin_idx == 0 ? num_rows : nulls_begin_idx;
    auto highest_timestamp_in_window = d_timestamps[idx] + following_window;

    return (thrust::upper_bound(thrust::seq,
                                d_timestamps + idx,
                                d_timestamps + group_end,
                                highest_timestamp_in_window) -
            (d_timestamps + idx)) -
           1;
  };

  auto following_column = expand_to_column(following_calculator, input.size(), stream, mr);

  return cudf::rolling_window(
    input, preceding_column->view(), following_column->view(), min_periods, aggr, mr);
}

/// Given a timestamp column grouped as specified in group_offsets,
/// return the following two vectors:
///  1. Vector with one entry per group, indicating the offset in the group
///     where the null values begin.
///  2. Vector with one entry per group, indicating the offset in the group
///     where the null values end. (i.e. 1 past the last null.)
/// Each group in the input timestamp column must be sorted,
/// with null values clustered at either the start or the end of each group.
/// If there are no nulls for any given group, (nulls_begin, nulls_end) == (0,0).
std::tuple<rmm::device_vector<size_type>, rmm::device_vector<size_type>>
get_null_bounds_for_timestamp_column(column_view const& timestamp_column,
                                     rmm::device_vector<size_type> const& group_offsets)
{
  // For each group, the null values are themselves clustered
  // at the beginning or the end of the group.
  // These nulls cannot participate, except in their own window.

  // If the input has n groups, group_offsets will have n+1 values.
  // null_start and null_end should eventually have 1 entry per group.
  auto null_start = rmm::device_vector<size_type>(group_offsets.begin(), group_offsets.end() - 1);
  auto null_end   = rmm::device_vector<size_type>(group_offsets.begin(), group_offsets.end() - 1);

  if (timestamp_column.has_nulls()) {
    auto p_timestamps_device_view = column_device_view::create(timestamp_column);
    auto num_groups               = group_offsets.size();

    // Null timestamps exist. Find null bounds, per group.
    thrust::for_each(
      thrust::device,
      thrust::make_counting_iterator(static_cast<size_type>(0)),
      thrust::make_counting_iterator(static_cast<size_type>(num_groups)),
      [d_timestamps    = *p_timestamps_device_view,
       d_group_offsets = group_offsets.data().get(),
       d_null_start    = null_start.data(),
       d_null_end      = null_end.data()] __device__(auto group_label) {
        auto group_start           = d_group_offsets[group_label];
        auto group_end             = d_group_offsets[group_label + 1];
        auto first_element_is_null = d_timestamps.is_null_nocheck(group_start);
        auto last_element_is_null  = d_timestamps.is_null_nocheck(group_end - 1);
        if (!first_element_is_null && !last_element_is_null) {
          // Short circuit: No nulls.
          d_null_start[group_label] = group_start;
          d_null_end[group_label]   = group_start;
        } else if (first_element_is_null && last_element_is_null) {
          // Short circuit: All nulls.
          d_null_start[group_label] = group_start;
          d_null_end[group_label]   = group_end;
        } else if (first_element_is_null) {
          // NULLS FIRST.
          d_null_start[group_label] = group_start;
          d_null_end[group_label]   = *thrust::partition_point(
            thrust::seq,
            thrust::make_counting_iterator(group_start),
            thrust::make_counting_iterator(group_end),
            [&d_timestamps] __device__(auto i) { return d_timestamps.is_null_nocheck(i); });
        } else {
          // NULLS LAST.
          d_null_end[group_label]   = group_end;
          d_null_start[group_label] = *thrust::partition_point(
            thrust::seq,
            thrust::make_counting_iterator(group_start),
            thrust::make_counting_iterator(group_end),
            [&d_timestamps] __device__(auto i) { return d_timestamps.is_valid_nocheck(i); });
        }
      });
  }

  return std::make_tuple(std::move(null_start), std::move(null_end));
}

// Time-range window computation, for timestamps in ASCENDING order.
std::unique_ptr<column> time_range_window_ASC(
  column_view const& input,
  column_view const& timestamp_column,
  rmm::device_vector<cudf::size_type> const& group_offsets,
  rmm::device_vector<cudf::size_type> const& group_labels,
  TimeT preceding_window,
  bool preceding_window_is_unbounded,
  TimeT following_window,
  bool following_window_is_unbounded,
  size_type min_periods,
  std::unique_ptr<aggregation> const& aggr,
  rmm::cuda_stream_view stream,
  rmm::mr::device_memory_resource* mr)
{
  rmm::device_vector<size_type> null_start, null_end;
  std::tie(null_start, null_end) =
    get_null_bounds_for_timestamp_column(timestamp_column, group_offsets);

  auto preceding_calculator =
    [d_group_offsets = group_offsets.data().get(),
     d_group_labels  = group_labels.data().get(),
     d_timestamps    = timestamp_column.data<TimeT>(),
     d_nulls_begin   = null_start.data().get(),
     d_nulls_end     = null_end.data().get(),
     preceding_window,
     preceding_window_is_unbounded] __device__(size_type idx) -> size_type {
    auto group_label = d_group_labels[idx];
    auto group_start = d_group_offsets[group_label];
    auto nulls_begin = d_nulls_begin[group_label];
    auto nulls_end   = d_nulls_end[group_label];

    if (preceding_window_is_unbounded) { return idx - group_start + 1; }

    // If idx lies in the null-range, the window is the null range.
    if (idx >= nulls_begin && idx < nulls_end) {
      // Current row is in the null group.
      // The window starts at the start of the null group.
      return idx - nulls_begin + 1;
    }

    // timestamp[idx] not null. Search must exclude the null group.
    // If nulls_begin == group_start, either of the following is true:
    //  1. NULLS FIRST ordering: Search must begin at nulls_end.
    //  2. NO NULLS: Search must begin at group_start (which also equals nulls_end.)
    // Otherwise, NULLS LAST ordering. Search must start at nulls group_start.
    auto search_start = nulls_begin == group_start ? nulls_end : group_start;

    auto lowest_timestamp_in_window = d_timestamps[idx] - preceding_window;

    return ((d_timestamps + idx) - thrust::lower_bound(thrust::seq,
                                                       d_timestamps + search_start,
                                                       d_timestamps + idx,
                                                       lowest_timestamp_in_window)) +
           1;  // Add 1, for `preceding` to account for current row.
  };

  auto preceding_column = expand_to_column(preceding_calculator, input.size(), stream, mr);

  auto following_calculator =
    [d_group_offsets = group_offsets.data().get(),
     d_group_labels  = group_labels.data().get(),
     d_timestamps    = timestamp_column.data<TimeT>(),
     d_nulls_begin   = null_start.data().get(),
     d_nulls_end     = null_end.data().get(),
     following_window,
     following_window_is_unbounded] __device__(size_type idx) -> size_type {
    auto group_label = d_group_labels[idx];
    auto group_start = d_group_offsets[group_label];
    auto group_end =
      d_group_offsets[group_label +
                      1];  // Cannot fall off the end, since offsets is capped with `input.size()`.
    auto nulls_begin = d_nulls_begin[group_label];
    auto nulls_end   = d_nulls_end[group_label];

    if (following_window_is_unbounded) { return (group_end - idx) - 1; }

    // If idx lies in the null-range, the window is the null range.
    if (idx >= nulls_begin && idx < nulls_end) {
      // Current row is in the null group.
      // The window ends at the end of the null group.
      return nulls_end - idx - 1;
    }

    // timestamp[idx] not null. Search must exclude the null group.
    // If nulls_begin == group_start, either of the following is true:
    //  1. NULLS FIRST ordering: Search ends at group_end.
    //  2. NO NULLS: Search ends at group_end.
    // Otherwise, NULLS LAST ordering. Search ends at nulls_begin.
    auto search_end = nulls_begin == group_start ? group_end : nulls_begin;

    auto highest_timestamp_in_window = d_timestamps[idx] + following_window;

    return (thrust::upper_bound(thrust::seq,
                                d_timestamps + idx,
                                d_timestamps + search_end,
                                highest_timestamp_in_window) -
            (d_timestamps + idx)) -
           1;
  };

  auto following_column = expand_to_column(following_calculator, input.size(), stream, mr);

  return cudf::rolling_window(
    input, preceding_column->view(), following_column->view(), min_periods, aggr, mr);
}

/// Time-range window computation, with
///   1. no grouping keys specified
///   2. timetamps in DESCENDING order.
/// Treat as one single group.
std::unique_ptr<column> time_range_window_DESC(column_view const& input,
                                               column_view const& timestamp_column,
                                               TimeT preceding_window,
                                               bool preceding_window_is_unbounded,
                                               TimeT following_window,
                                               bool following_window_is_unbounded,
                                               size_type min_periods,
                                               std::unique_ptr<aggregation> const& aggr,
                                               rmm::cuda_stream_view stream,
                                               rmm::mr::device_memory_resource* mr)
{
  size_type nulls_begin_idx, nulls_end_idx;
  std::tie(nulls_begin_idx, nulls_end_idx) = get_null_bounds_for_timestamp_column(timestamp_column);

  auto preceding_calculator =
    [nulls_begin_idx,
     nulls_end_idx,
     d_timestamps = timestamp_column.data<TimeT>(),
     preceding_window,
     preceding_window_is_unbounded] __device__(size_type idx) -> size_type {
    if (preceding_window_is_unbounded) {
      return idx + 1;  // Technically `idx - 0 + 1`,
                       // where 0 == Group start,
                       // and   1 accounts for the current row
    }
    if (idx >= nulls_begin_idx && idx < nulls_end_idx) {
      // Current row is in the null group.
      // Must consider beginning of null-group as window start.
      return idx - nulls_begin_idx + 1;
    }

    // timestamp[idx] not null. Binary search the group, excluding null group.
    // If nulls_begin_idx == 0, either
    //  1. NULLS FIRST ordering: Binary search starts where nulls_end_idx.
    //  2. NO NULLS: Binary search starts at 0 (also nulls_end_idx).
    // Otherwise, NULLS LAST ordering. Start at 0.
    auto group_start                 = nulls_begin_idx == 0 ? nulls_end_idx : 0;
    auto highest_timestamp_in_window = d_timestamps[idx] + preceding_window;

    return ((d_timestamps + idx) -
            thrust::lower_bound(thrust::seq,
                                d_timestamps + group_start,
                                d_timestamps + idx,
                                highest_timestamp_in_window,
                                thrust::greater<decltype(highest_timestamp_in_window)>())) +
           1;  // Add 1, for `preceding` to account for current row.
  };

  auto preceding_column = expand_to_column(preceding_calculator, input.size(), stream, mr);

  auto following_calculator =
    [nulls_begin_idx,
     nulls_end_idx,
     num_rows     = input.size(),
     d_timestamps = timestamp_column.data<TimeT>(),
     following_window,
     following_window_is_unbounded] __device__(size_type idx) -> size_type {
    if (following_window_is_unbounded) { return (num_rows - idx) - 1; }
    if (idx >= nulls_begin_idx && idx < nulls_end_idx) {
      // Current row is in the null group.
      // Window ends at the end of the null group.
      return nulls_end_idx - idx - 1;
    }

    // timestamp[idx] not null. Search must exclude null group.
    // If nulls_begin_idx = 0, either
    //  1. NULLS FIRST ordering: Search ends at num_rows.
    //  2. NO NULLS: Search also ends at num_rows.
    // Otherwise, NULLS LAST ordering: End at nulls_begin_idx.

    auto group_end                  = nulls_begin_idx == 0 ? num_rows : nulls_begin_idx;
    auto lowest_timestamp_in_window = d_timestamps[idx] - following_window;

    return (thrust::upper_bound(thrust::seq,
                                d_timestamps + idx,
                                d_timestamps + group_end,
                                lowest_timestamp_in_window,
                                thrust::greater<decltype(lowest_timestamp_in_window)>()) -
            (d_timestamps + idx)) -
           1;
  };

  auto following_column = expand_to_column(following_calculator, input.size(), stream, mr);

  return cudf::rolling_window(
    input, preceding_column->view(), following_column->view(), min_periods, aggr, mr);
}

// Time-range window computation, for timestamps in DESCENDING order.
std::unique_ptr<column> time_range_window_DESC(
  column_view const& input,
  column_view const& timestamp_column,
  rmm::device_vector<cudf::size_type> const& group_offsets,
  rmm::device_vector<cudf::size_type> const& group_labels,
  TimeT preceding_window,
  bool preceding_window_is_unbounded,
  TimeT following_window,
  bool following_window_is_unbounded,
  size_type min_periods,
  std::unique_ptr<aggregation> const& aggr,
  rmm::cuda_stream_view stream,
  rmm::mr::device_memory_resource* mr)
{
  rmm::device_vector<size_type> null_start, null_end;
  std::tie(null_start, null_end) =
    get_null_bounds_for_timestamp_column(timestamp_column, group_offsets);

  auto preceding_calculator =
    [d_group_offsets = group_offsets.data().get(),
     d_group_labels  = group_labels.data().get(),
     d_timestamps    = timestamp_column.data<TimeT>(),
     d_nulls_begin   = null_start.data().get(),
     d_nulls_end     = null_end.data().get(),
     preceding_window,
     preceding_window_is_unbounded] __device__(size_type idx) -> size_type {
    auto group_label = d_group_labels[idx];
    auto group_start = d_group_offsets[group_label];
    auto nulls_begin = d_nulls_begin[group_label];
    auto nulls_end   = d_nulls_end[group_label];

    if (preceding_window_is_unbounded) { return (idx - group_start) + 1; }

    // If idx lies in the null-range, the window is the null range.
    if (idx >= nulls_begin && idx < nulls_end) {
      // Current row is in the null group.
      // The window starts at the start of the null group.
      return idx - nulls_begin + 1;
    }

    // timestamp[idx] not null. Search must exclude the null group.
    // If nulls_begin == group_start, either of the following is true:
    //  1. NULLS FIRST ordering: Search must begin at nulls_end.
    //  2. NO NULLS: Search must begin at group_start (which also equals nulls_end.)
    // Otherwise, NULLS LAST ordering. Search must start at nulls group_start.
    auto search_start = nulls_begin == group_start ? nulls_end : group_start;

    auto highest_timestamp_in_window = d_timestamps[idx] + preceding_window;

    return ((d_timestamps + idx) -
            thrust::lower_bound(thrust::seq,
                                d_timestamps + search_start,
                                d_timestamps + idx,
                                highest_timestamp_in_window,
                                thrust::greater<decltype(highest_timestamp_in_window)>())) +
           1;  // Add 1, for `preceding` to account for current row.
  };

  auto preceding_column = expand_to_column(preceding_calculator, input.size(), stream, mr);

  auto following_calculator =
    [d_group_offsets = group_offsets.data().get(),
     d_group_labels  = group_labels.data().get(),
     d_timestamps    = timestamp_column.data<TimeT>(),
     d_nulls_begin   = null_start.data().get(),
     d_nulls_end     = null_end.data().get(),
     following_window,
     following_window_is_unbounded] __device__(size_type idx) -> size_type {
    auto group_label = d_group_labels[idx];
    auto group_start = d_group_offsets[group_label];
    auto group_end   = d_group_offsets[group_label + 1];
    auto nulls_begin = d_nulls_begin[group_label];
    auto nulls_end   = d_nulls_end[group_label];

    if (following_window_is_unbounded) { return (group_end - idx) - 1; }

    // If idx lies in the null-range, the window is the null range.
    if (idx >= nulls_begin && idx < nulls_end) {
      // Current row is in the null group.
      // The window ends at the end of the null group.
      return nulls_end - idx - 1;
    }

    // timestamp[idx] not null. Search must exclude the null group.
    // If nulls_begin == group_start, either of the following is true:
    //  1. NULLS FIRST ordering: Search ends at group_end.
    //  2. NO NULLS: Search ends at group_end.
    // Otherwise, NULLS LAST ordering. Search ends at nulls_begin.
    auto search_end = nulls_begin == group_start ? group_end : nulls_begin;

    auto lowest_timestamp_in_window = d_timestamps[idx] - following_window;

    return (thrust::upper_bound(thrust::seq,
                                d_timestamps + idx,
                                d_timestamps + search_end,
                                lowest_timestamp_in_window,
                                thrust::greater<decltype(lowest_timestamp_in_window)>()) -
            (d_timestamps + idx)) -
           1;
  };

  auto following_column = expand_to_column(following_calculator, input.size(), stream, mr);

  if (aggr->kind == aggregation::CUDA || aggr->kind == aggregation::PTX) {
    CUDF_FAIL("Time ranged rolling window does NOT (yet) support UDF.");
  } else {
    return cudf::rolling_window(
      input, preceding_column->view(), following_column->view(), min_periods, aggr, mr);
  }
}

std::unique_ptr<column> grouped_time_range_rolling_window_impl(
  column_view const& input,
  column_view const& timestamp_column,
  cudf::order const& timestamp_ordering,
  rmm::device_vector<cudf::size_type> const& group_offsets,
  rmm::device_vector<cudf::size_type> const& group_labels,
  window_bounds preceding_window_in_days,  // TODO: Consider taking offset-type as type_id. Assumes
                                           // days for now.
  window_bounds following_window_in_days,
  size_type min_periods,
  std::unique_ptr<aggregation> const& aggr,
  rmm::cuda_stream_view stream,
  rmm::mr::device_memory_resource* mr)
{
  TimeT mult_factor{static_cast<TimeT>(multiplication_factor(timestamp_column.type()))};

  if (timestamp_ordering == cudf::order::ASCENDING) {
    return group_offsets.empty()
             ? time_range_window_ASC(input,
                                     timestamp_column,
                                     preceding_window_in_days.value * mult_factor,
                                     preceding_window_in_days.is_unbounded,
                                     following_window_in_days.value * mult_factor,
                                     following_window_in_days.is_unbounded,
                                     min_periods,
                                     aggr,
                                     stream,
                                     mr)
             : time_range_window_ASC(input,
                                     timestamp_column,
                                     group_offsets,
                                     group_labels,
                                     preceding_window_in_days.value * mult_factor,
                                     preceding_window_in_days.is_unbounded,
                                     following_window_in_days.value * mult_factor,
                                     following_window_in_days.is_unbounded,
                                     min_periods,
                                     aggr,
                                     stream,
                                     mr);
  } else {
    return group_offsets.empty()
             ? time_range_window_DESC(input,
                                      timestamp_column,
                                      preceding_window_in_days.value * mult_factor,
                                      preceding_window_in_days.is_unbounded,
                                      following_window_in_days.value * mult_factor,
                                      following_window_in_days.is_unbounded,
                                      min_periods,
                                      aggr,
                                      stream,
                                      mr)
             : time_range_window_DESC(input,
                                      timestamp_column,
                                      group_offsets,
                                      group_labels,
                                      preceding_window_in_days.value * mult_factor,
                                      preceding_window_in_days.is_unbounded,
                                      following_window_in_days.value * mult_factor,
                                      following_window_in_days.is_unbounded,
                                      min_periods,
                                      aggr,
                                      stream,
                                      mr);
  }
}

}  // namespace

namespace detail {

std::unique_ptr<column> grouped_time_range_rolling_window(table_view const& group_keys,
                                                          column_view const& timestamp_column,
                                                          cudf::order const& timestamp_order,
                                                          column_view const& input,
                                                          window_bounds preceding_window_in_days,
                                                          window_bounds following_window_in_days,
                                                          size_type min_periods,
                                                          std::unique_ptr<aggregation> const& aggr,
                                                          rmm::cuda_stream_view stream,
                                                          rmm::mr::device_memory_resource* mr)
{
  CUDF_FUNC_RANGE();

  if (input.is_empty()) return empty_like(input);

  CUDF_EXPECTS((group_keys.num_columns() == 0 || group_keys.num_rows() == input.size()),
               "Size mismatch between group_keys and input vector.");

  CUDF_EXPECTS((min_periods > 0), "min_periods must be positive");

  using sort_groupby_helper = cudf::groupby::detail::sort::sort_groupby_helper;
  using index_vector        = sort_groupby_helper::index_vector;

  index_vector group_offsets, group_labels;
  if (group_keys.num_columns() > 0) {
    sort_groupby_helper helper{group_keys, cudf::null_policy::INCLUDE, cudf::sorted::YES};
    group_offsets = helper.group_offsets();
    group_labels  = helper.group_labels();
  }

  // Assumes that `timestamp_column` is actually of a timestamp type.
  CUDF_EXPECTS(is_supported_range_frame_unit(timestamp_column.type()),
               "Unsupported data-type for `timestamp`-based rolling window operation!");

  auto is_timestamp_in_days = timestamp_column.type().id() == cudf::type_id::TIMESTAMP_DAYS;

  return grouped_time_range_rolling_window_impl(
    input,
    is_timestamp_in_days
      ? cudf::cast(timestamp_column, cudf::data_type(cudf::type_id::TIMESTAMP_SECONDS), mr)->view()
      : timestamp_column,
    timestamp_order,
    group_offsets,
    group_labels,
    preceding_window_in_days,
    following_window_in_days,
    min_periods,
    aggr,
    stream,
    mr);
}

}  // namespace detail

std::unique_ptr<column> grouped_time_range_rolling_window(table_view const& group_keys,
                                                          column_view const& timestamp_column,
                                                          cudf::order const& timestamp_order,
                                                          column_view const& input,
                                                          size_type preceding_window_in_days,
                                                          size_type following_window_in_days,
                                                          size_type min_periods,
                                                          std::unique_ptr<aggregation> const& aggr,
                                                          rmm::mr::device_memory_resource* mr)
{
  return grouped_time_range_rolling_window(group_keys,
                                           timestamp_column,
                                           timestamp_order,
                                           input,
                                           window_bounds::get(preceding_window_in_days),
                                           window_bounds::get(following_window_in_days),
                                           min_periods,
                                           aggr,
                                           rmm::cuda_stream_default,
                                           mr);
}

std::unique_ptr<column> grouped_time_range_rolling_window(table_view const& group_keys,
                                                          column_view const& timestamp_column,
                                                          cudf::order const& timestamp_order,
                                                          column_view const& input,
                                                          window_bounds preceding_window_in_days,
                                                          window_bounds following_window_in_days,
                                                          size_type min_periods,
                                                          std::unique_ptr<aggregation> const& aggr,
                                                          rmm::mr::device_memory_resource* mr)
{
  return detail::grouped_time_range_rolling_window(group_keys,
                                                   timestamp_column,
                                                   timestamp_order,
                                                   input,
                                                   preceding_window_in_days,
                                                   following_window_in_days,
                                                   min_periods,
                                                   aggr,
                                                   rmm::cuda_stream_default,
                                                   mr);
}

}  // namespace cudf
