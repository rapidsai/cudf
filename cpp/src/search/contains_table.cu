#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2022-2023, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <join/join_common_utils.cuh>

#include <cudf/detail/join.hpp>
#include <cudf/detail/null_mask.hpp>
#include <cudf/detail/structs/utilities.hpp>
#include <cudf/table/experimental/row_operators.cuh>
#include <cudf/table/table_device_view.cuh>
#include <cudf/table/table_view.hpp>
#include <cudf/types.hpp>

#include <rmm/cuda_stream_view.hpp>
#include <rmm/device_uvector.hpp>

#include <thrust/iterator/counting_iterator.h>

#include <cuco/static_map.cuh>

#include <type_traits>

namespace cudf::detail {

namespace {

using cudf::experimental::row::lhs_index_type;
using cudf::experimental::row::rhs_index_type;

using static_map = cuco::static_map<lhs_index_type,
                                    size_type,
                                    cuda::thread_scope_device,
                                    rmm::mr::stream_allocator_adaptor<default_allocator<char>>>;

/**
 * @brief Check if the given type `T` is a strong index type (i.e., `lhs_index_type` or
 * `rhs_index_type`).
 *
 * @return A boolean value indicating if `T` is a strong index type
 */
template <typename T>
constexpr auto is_strong_index_type()
{
  return std::is_same_v<T, lhs_index_type> || std::is_same_v<T, rhs_index_type>;
}

/**
 * @brief An adapter functor to support strong index types for row hasher that must be operating on
 * `cudf::size_type`.
 */
template <typename Hasher>
struct strong_index_hasher_adapter {
  strong_index_hasher_adapter(Hasher const& hasher) : _hasher{hasher} {}

  template <typename T, CUDF_ENABLE_IF(is_strong_index_type<T>())>
  __device__ constexpr auto operator()(T const idx) const noexcept
  {
    return _hasher(static_cast<size_type>(idx));
  }

 private:
  Hasher const _hasher;
};

/**
 * @brief An adapter functor to support strong index type for table row comparator that must be
 * operating on `cudf::size_type`.
 */
template <typename Comparator>
struct strong_index_comparator_adapter {
  strong_index_comparator_adapter(Comparator const& comparator) : _comparator{comparator} {}

  template <typename T,
            typename U,
            CUDF_ENABLE_IF(is_strong_index_type<T>() && is_strong_index_type<U>())>
  __device__ constexpr auto operator()(T const lhs_index, U const rhs_index) const noexcept
  {
    auto const lhs = static_cast<size_type>(lhs_index);
    auto const rhs = static_cast<size_type>(rhs_index);

    if constexpr (std::is_same_v<T, U> || std::is_same_v<T, lhs_index_type>) {
      return _comparator(lhs, rhs);
    } else {
      // Here we have T == rhs_index_type.
      // This is when the indices are provided in wrong order for two table comparator, so we need
      // to switch them back to the right order before calling the underlying comparator.
      return _comparator(rhs, lhs);
    }
  }

 private:
  Comparator const _comparator;
};

/**
 * @brief Build a row bitmask for the input table.
 *
 * The output bitmask will have invalid bits corresponding to the the input rows having nulls (at
 * any nested level) and vice versa.
 *
 * @param input The input table
 * @param stream CUDA stream used for device memory operations and kernel launches
 * @return A pair of pointer to the output bitmask and the buffer containing the bitmask
 */
std::pair<rmm::device_buffer, bitmask_type const*> build_row_bitmask(table_view const& input,
                                                                     rmm::cuda_stream_view stream)
{
  auto const nullable_columns = get_nullable_columns(input);
  CUDF_EXPECTS(nullable_columns.size() > 0,
               "The input table has nulls thus it should have nullable columns.");

  // If there are more than one nullable column, we compute `bitmask_and` of their null masks.
  // Otherwise, we have only one nullable column and can use its null mask directly.
  if (nullable_columns.size() > 1) {
    auto row_bitmask = cudf::detail::bitmask_and(table_view{nullable_columns}, stream).first;
    auto const row_bitmask_ptr = static_cast<bitmask_type const*>(row_bitmask.data());
    return std::pair(std::move(row_bitmask), row_bitmask_ptr);
  }

  return std::pair(rmm::device_buffer{0, stream}, nullable_columns.front().null_mask());
}

/**
 * @brief Invoke an `operator()` template with a row equality comparator based on the specified
 * `compare_nans` parameter.
 *
 * @param compare_nans The flag to specify whether NaNs should be compared equal or not
 * @param func The input functor to invoke
 */
template <typename Func>
void dispatch_nan_comparator(nan_equality compare_nans, Func&& func)
{
  if (compare_nans == nan_equality::ALL_EQUAL) {
    using nan_equal_comparator =
      cudf::experimental::row::equality::nan_equal_physical_equality_comparator;
    func(nan_equal_comparator{});
  } else {
    using nan_unequal_comparator = cudf::experimental::row::equality::physical_equality_comparator;
    func(nan_unequal_comparator{});
  }
}

/**
 * @brief Check if rows in the given `needles` table exist in the `haystack` table.
 *
 * This function is designed specifically to work with input tables having lists column(s) at
 * arbitrarily nested levels.
 *
 * @param haystack The table containing the search space
 * @param needles A table of rows whose existence to check in the search space
 * @param compare_nulls Control whether nulls should be compared as equal or not
 * @param compare_nans Control whether floating-point NaNs values should be compared as equal or not
 * @param stream CUDA stream used for device memory operations and kernel launches
 * @param mr Device memory resource used to allocate the returned vector
 * @return A vector of bools indicating if each row in `needles` has matching rows in `haystack`
 */
rmm::device_uvector<bool> contains_with_lists_or_nans(table_view const& haystack,
                                                      table_view const& needles,
                                                      null_equality compare_nulls,
                                                      nan_equality compare_nans,
                                                      rmm::cuda_stream_view stream,
                                                      rmm::mr::device_memory_resource* mr)
{
  auto map = static_map(compute_hash_table_size(haystack.num_rows()),
                        cuco::empty_key{lhs_index_type{std::numeric_limits<size_type>::max()}},
                        cuco::empty_value{detail::JoinNoneValue},
                        detail::hash_table_allocator_type{default_allocator<char>{}, stream},
                        stream.value());

  auto const haystack_has_nulls = has_nested_nulls(haystack);
  auto const needles_has_nulls  = has_nested_nulls(needles);
  auto const has_any_nulls      = haystack_has_nulls || needles_has_nulls;

  // Insert row indices of the haystack table as map keys.
  {
    auto const haystack_it = cudf::detail::make_counting_transform_iterator(
      size_type{0},
      [] __device__(auto const idx) { return cuco::make_pair(lhs_index_type{idx}, 0); });

    auto const hasher = cudf::experimental::row::hash::row_hasher(haystack, stream);
    auto const d_hasher =
      strong_index_hasher_adapter{hasher.device_hasher(nullate::DYNAMIC{has_any_nulls})};

    auto const comparator = cudf::experimental::row::equality::self_comparator(haystack, stream);

    // If the haystack table has nulls but they are compared unequal, don't insert them.
    // Otherwise, it was known to cause performance issue:
    // - https://github.com/rapidsai/cudf/pull/6943
    // - https://github.com/rapidsai/cudf/pull/8277
    if (haystack_has_nulls && compare_nulls == null_equality::UNEQUAL) {
      auto const bitmask_buffer_and_ptr = build_row_bitmask(haystack, stream);
      auto const row_bitmask_ptr        = bitmask_buffer_and_ptr.second;

      auto const insert_map = [&](auto const value_comp) {
        if (cudf::detail::has_nested_columns(haystack)) {
          auto const d_eqcomp = strong_index_comparator_adapter{comparator.equal_to<true>(
            nullate::DYNAMIC{haystack_has_nulls}, compare_nulls, value_comp)};
          map.insert_if(haystack_it,
                        haystack_it + haystack.num_rows(),
                        thrust::counting_iterator<size_type>(0),  // stencil
                        row_is_valid{row_bitmask_ptr},
                        d_hasher,
                        d_eqcomp,
                        stream.value());
        } else {
          auto const d_eqcomp = strong_index_comparator_adapter{comparator.equal_to<false>(
            nullate::DYNAMIC{haystack_has_nulls}, compare_nulls, value_comp)};
          map.insert_if(haystack_it,
                        haystack_it + haystack.num_rows(),
                        thrust::counting_iterator<size_type>(0),  // stencil
                        row_is_valid{row_bitmask_ptr},
                        d_hasher,
                        d_eqcomp,
                        stream.value());
        }
      };

      // Insert only rows that do not have any null at any level.
      dispatch_nan_comparator(compare_nans, insert_map);
    } else {  // haystack_doesn't_have_nulls || compare_nulls == null_equality::EQUAL
      auto const insert_map = [&](auto const value_comp) {
        if (cudf::detail::has_nested_columns(haystack)) {
          auto const d_eqcomp = strong_index_comparator_adapter{comparator.equal_to<true>(
            nullate::DYNAMIC{haystack_has_nulls}, compare_nulls, value_comp)};
          map.insert(
            haystack_it, haystack_it + haystack.num_rows(), d_hasher, d_eqcomp, stream.value());
        } else {
          auto const d_eqcomp = strong_index_comparator_adapter{comparator.equal_to<false>(
            nullate::DYNAMIC{haystack_has_nulls}, compare_nulls, value_comp)};
          map.insert(
            haystack_it, haystack_it + haystack.num_rows(), d_hasher, d_eqcomp, stream.value());
        }
      };

      dispatch_nan_comparator(compare_nans, insert_map);
    }
  }

  // The output vector.
  auto contained = rmm::device_uvector<bool>(needles.num_rows(), stream, mr);

  // Check existence for each row of the needles table in the haystack table.
  {
    auto const needles_it = cudf::detail::make_counting_transform_iterator(
      size_type{0}, [] __device__(auto const idx) { return rhs_index_type{idx}; });

    auto const hasher = cudf::experimental::row::hash::row_hasher(needles, stream);
    auto const d_hasher =
      strong_index_hasher_adapter{hasher.device_hasher(nullate::DYNAMIC{has_any_nulls})};

    auto const comparator =
      cudf::experimental::row::equality::two_table_comparator(haystack, needles, stream);

    auto const check_contains = [&](auto const value_comp) {
      if (cudf::detail::has_nested_columns(haystack) or cudf::detail::has_nested_columns(needles)) {
        auto const d_eqcomp =
          comparator.equal_to<true>(nullate::DYNAMIC{has_any_nulls}, compare_nulls, value_comp);
        map.contains(needles_it,
                     needles_it + needles.num_rows(),
                     contained.begin(),
                     d_hasher,
                     d_eqcomp,
                     stream.value());
      } else {
        auto const d_eqcomp =
          comparator.equal_to<false>(nullate::DYNAMIC{has_any_nulls}, compare_nulls, value_comp);
        map.contains(needles_it,
                     needles_it + needles.num_rows(),
                     contained.begin(),
                     d_hasher,
                     d_eqcomp,
                     stream.value());
      }
    };

    dispatch_nan_comparator(compare_nans, check_contains);
  }

  return contained;
}

/**
 * @brief Check if rows in the given `needles` table exist in the `haystack` table.
 *
 * This function is designed specifically to work with input tables having only columns of simple
 * types, or structs columns of simple types.
 *
 * @param haystack The table containing the search space
 * @param needles A table of rows whose existence to check in the search space
 * @param compare_nulls Control whether nulls should be compared as equal or not
 * @param stream CUDA stream used for device memory operations and kernel launches
 * @param mr Device memory resource used to allocate the returned vector
 * @return A vector of bools indicating if each row in `needles` has matching rows in `haystack`
 */
rmm::device_uvector<bool> contains_without_lists_or_nans(table_view const& haystack,
                                                         table_view const& needles,
                                                         null_equality compare_nulls,
                                                         rmm::cuda_stream_view stream,
                                                         rmm::mr::device_memory_resource* mr)
{
  auto map = static_map(compute_hash_table_size(haystack.num_rows()),
                        cuco::empty_key{lhs_index_type{std::numeric_limits<size_type>::max()}},
                        cuco::empty_value{detail::JoinNoneValue},
                        detail::hash_table_allocator_type{default_allocator<char>{}, stream},
                        stream.value());

  auto const haystack_has_nulls = has_nested_nulls(haystack);
  auto const needles_has_nulls  = has_nested_nulls(needles);
  auto const has_any_nulls      = haystack_has_nulls || needles_has_nulls;

  // Flatten the input tables.
  auto const flatten_nullability = has_any_nulls
                                     ? structs::detail::column_nullability::FORCE
                                     : structs::detail::column_nullability::MATCH_INCOMING;
  auto const haystack_flattened_tables =
    structs::detail::flatten_nested_columns(haystack, {}, {}, flatten_nullability, stream);
  auto const needles_flattened_tables =
    structs::detail::flatten_nested_columns(needles, {}, {}, flatten_nullability, stream);
  auto const haystack_flattened = haystack_flattened_tables->flattened_columns();
  auto const needles_flattened  = needles_flattened_tables->flattened_columns();
  auto const haystack_tdv_ptr   = table_device_view::create(haystack_flattened, stream);
  auto const needles_tdv_ptr    = table_device_view::create(needles_flattened, stream);

  // Insert row indices of the haystack table as map keys.
  {
    auto const haystack_it = cudf::detail::make_counting_transform_iterator(
      size_type{0},
      [] __device__(auto const idx) { return cuco::make_pair(lhs_index_type{idx}, 0); });

    auto const d_hasher = strong_index_hasher_adapter{
      row_hash{cudf::nullate::DYNAMIC{has_any_nulls}, *haystack_tdv_ptr}};
    auto const d_eqcomp =
      strong_index_comparator_adapter{row_equality{cudf::nullate::DYNAMIC{haystack_has_nulls},
                                                   *haystack_tdv_ptr,
                                                   *haystack_tdv_ptr,
                                                   compare_nulls}};

    // If the haystack table has nulls but they are compared unequal, don't insert them.
    // Otherwise, it was known to cause performance issue:
    // - https://github.com/rapidsai/cudf/pull/6943
    // - https://github.com/rapidsai/cudf/pull/8277
    if (haystack_has_nulls && compare_nulls == null_equality::UNEQUAL) {
      auto const bitmask_buffer_and_ptr = build_row_bitmask(haystack, stream);
      auto const row_bitmask_ptr        = bitmask_buffer_and_ptr.second;

      // Insert only rows that do not have any null at any level.
      map.insert_if(haystack_it,
                    haystack_it + haystack.num_rows(),
                    thrust::counting_iterator<size_type>(0),  // stencil
                    row_is_valid{row_bitmask_ptr},
                    d_hasher,
                    d_eqcomp,
                    stream.value());

    } else {  // haystack_doesn't_have_nulls || compare_nulls == null_equality::EQUAL
      map.insert(
        haystack_it, haystack_it + haystack.num_rows(), d_hasher, d_eqcomp, stream.value());
    }
  }

  // The output vector.
  auto contained = rmm::device_uvector<bool>(needles.num_rows(), stream, mr);

  // Check existence for each row of the needles table in the haystack table.
  {
    auto const needles_it = cudf::detail::make_counting_transform_iterator(
      size_type{0}, [] __device__(auto const idx) { return rhs_index_type{idx}; });

    auto const d_hasher = strong_index_hasher_adapter{
      row_hash{cudf::nullate::DYNAMIC{has_any_nulls}, *needles_tdv_ptr}};

    auto const d_eqcomp = strong_index_comparator_adapter{row_equality{
      cudf::nullate::DYNAMIC{has_any_nulls}, *haystack_tdv_ptr, *needles_tdv_ptr, compare_nulls}};

    map.contains(needles_it,
                 needles_it + needles.num_rows(),
                 contained.begin(),
                 d_hasher,
                 d_eqcomp,
                 stream.value());
  }

  return contained;
}

}  // namespace

rmm::device_uvector<bool> contains(table_view const& haystack,
                                   table_view const& needles,
                                   null_equality compare_nulls,
                                   nan_equality compare_nans,
                                   rmm::cuda_stream_view stream,
                                   rmm::mr::device_memory_resource* mr)
{
  // Checking for only one table is enough, because both tables will be checked to have the same
  // shape later during row comparisons.
  auto const has_lists = std::any_of(haystack.begin(), haystack.end(), [](auto const& col) {
    return cudf::structs::detail::is_or_has_nested_lists(col);
  });

  if (has_lists || compare_nans == nan_equality::UNEQUAL) {
    // We must call a separate code path that uses the new experimental row hasher and row
    // comparator if:
    //  - The input has lists column, or
    //  - Floating-point NaNs are compared as unequal.
    // Inputs with these conditions are supported only by this code path.
    return contains_with_lists_or_nans(haystack, needles, compare_nulls, compare_nans, stream, mr);
  }

  // If the input tables don't have lists column and NaNs are compared equal, we rely on the classic
  // code path that flattens the input tables for row comparisons. This way is known to have
  // better performance.
  return contains_without_lists_or_nans(haystack, needles, compare_nulls, stream, mr);

  // Note: We have to keep separate code paths because unifying them will cause performance
  // regression for the input having no nested lists.
  //
  // TODO: We should unify these code paths in the future when performance regression is no longer
  // happening.
}

}  // namespace cudf::detail
