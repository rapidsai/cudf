/*
 * Copyright (c) 2019-2020, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <cudf/column/column_factories.hpp>
#include <cudf/detail/iterator.cuh>
#include <cudf/detail/nvtx/ranges.hpp>
#include <cudf/detail/search.hpp>
#include <cudf/dictionary/detail/search.hpp>
#include <cudf/dictionary/detail/update_keys.hpp>
#include <cudf/scalar/scalar_device_view.cuh>
#include <cudf/search.hpp>
#include <cudf/table/row_operators.cuh>
#include <cudf/table/table_device_view.cuh>
#include <cudf/table/table_view.hpp>

#include <hash/unordered_multiset.cuh>

#include <rmm/cuda_stream_view.hpp>

#include <thrust/binary_search.h>

namespace cudf {
namespace {
template <typename DataIterator,
          typename ValuesIterator,
          typename OutputIterator,
          typename Comparator>
void launch_search(DataIterator it_data,
                   ValuesIterator it_vals,
                   size_type data_size,
                   size_type values_size,
                   OutputIterator it_output,
                   Comparator comp,
                   bool find_first,
                   rmm::cuda_stream_view stream)
{
  if (find_first) {
    thrust::lower_bound(rmm::exec_policy(stream)->on(stream.value()),
                        it_data,
                        it_data + data_size,
                        it_vals,
                        it_vals + values_size,
                        it_output,
                        comp);
  } else {
    thrust::upper_bound(rmm::exec_policy(stream)->on(stream.value()),
                        it_data,
                        it_data + data_size,
                        it_vals,
                        it_vals + values_size,
                        it_output,
                        comp);
  }
}

std::unique_ptr<column> search_ordered(table_view const& t,
                                       table_view const& values,
                                       bool find_first,
                                       std::vector<order> const& column_order,
                                       std::vector<null_order> const& null_precedence,
                                       rmm::cuda_stream_view stream,
                                       rmm::mr::device_memory_resource* mr)
{
  // Allocate result column
  std::unique_ptr<column> result = make_numeric_column(
    data_type{type_to_id<size_type>()}, values.num_rows(), mask_state::UNALLOCATED, stream, mr);

  mutable_column_view result_view = result.get()->mutable_view();

  // Handle empty inputs
  if (t.num_rows() == 0) {
    CUDA_TRY(hipMemsetAsync(
      result_view.data<size_type>(), 0, values.num_rows() * sizeof(size_type), stream.value()));
    return result;
  }

  if (not column_order.empty()) {
    CUDF_EXPECTS(static_cast<std::size_t>(t.num_columns()) == column_order.size(),
                 "Mismatch between number of columns and column order.");
  }

  if (not null_precedence.empty()) {
    CUDF_EXPECTS(static_cast<std::size_t>(t.num_columns()) == null_precedence.size(),
                 "Mismatch between number of columns and null precedence.");
  }

  // This utility will ensure all corresponding dictionary columns have matching keys.
  // It will return any new dictionary columns created as well as updated table_views.
  auto matched  = dictionary::detail::match_dictionaries({t, values}, stream);
  auto d_t      = table_device_view::create(matched.second.front(), stream);
  auto d_values = table_device_view::create(matched.second.back(), stream);
  auto count_it = thrust::make_counting_iterator<size_type>(0);

  rmm::device_vector<order> d_column_order(column_order.begin(), column_order.end());
  rmm::device_vector<null_order> d_null_precedence(null_precedence.begin(), null_precedence.end());

  if (has_nulls(t) or has_nulls(values)) {
    auto ineq_op =
      (find_first)
        ? row_lexicographic_comparator<true>(
            *d_t, *d_values, d_column_order.data().get(), d_null_precedence.data().get())
        : row_lexicographic_comparator<true>(
            *d_values, *d_t, d_column_order.data().get(), d_null_precedence.data().get());

    launch_search(count_it,
                  count_it,
                  t.num_rows(),
                  values.num_rows(),
                  result_view.data<size_type>(),
                  ineq_op,
                  find_first,
                  stream);
  } else {
    auto ineq_op =
      (find_first)
        ? row_lexicographic_comparator<false>(
            *d_t, *d_values, d_column_order.data().get(), d_null_precedence.data().get())
        : row_lexicographic_comparator<false>(
            *d_values, *d_t, d_column_order.data().get(), d_null_precedence.data().get());

    launch_search(count_it,
                  count_it,
                  t.num_rows(),
                  values.num_rows(),
                  result_view.data<size_type>(),
                  ineq_op,
                  find_first,
                  stream);
  }

  return result;
}

struct contains_scalar_dispatch {
  template <typename Element>
  bool operator()(column_view const& col, scalar const& value, rmm::cuda_stream_view stream)
  {
    CUDF_EXPECTS(col.type() == value.type(), "scalar and column types must match");

    using Type       = device_storage_type_t<Element>;
    using ScalarType = cudf::scalar_type_t<Element>;
    auto d_col       = column_device_view::create(col, stream);
    auto s           = static_cast<const ScalarType*>(&value);

    if (col.has_nulls()) {
      auto found_iter = thrust::find(rmm::exec_policy(stream)->on(stream.value()),
                                     d_col->pair_begin<Type, true>(),
                                     d_col->pair_end<Type, true>(),
                                     thrust::make_pair(s->value(), true));

      return found_iter != d_col->pair_end<Type, true>();
    } else {
      auto found_iter = thrust::find(rmm::exec_policy(stream)->on(stream.value()),  //
                                     d_col->begin<Type>(),
                                     d_col->end<Type>(),
                                     s->value());

      return found_iter != d_col->end<Type>();
    }
  }
};

template <>
bool contains_scalar_dispatch::operator()<cudf::list_view>(column_view const& col,
                                                           scalar const& value,
                                                           rmm::cuda_stream_view stream)
{
  CUDF_FAIL("list_view type not supported yet");
}

template <>
bool contains_scalar_dispatch::operator()<cudf::struct_view>(column_view const& col,
                                                             scalar const& value,
                                                             rmm::cuda_stream_view stream)
{
  CUDF_FAIL("struct_view type not supported yet");
}

template <>
bool contains_scalar_dispatch::operator()<cudf::dictionary32>(column_view const& col,
                                                              scalar const& value,
                                                              rmm::cuda_stream_view stream)
{
  auto dict_col = cudf::dictionary_column_view(col);
  // first, find the value in the dictionary's key set
  auto index = cudf::dictionary::detail::get_index(dict_col, value, stream);
  // if found, check the index is actually in the indices column
  return index->is_valid() ? cudf::type_dispatcher(dict_col.indices().type(),
                                                   contains_scalar_dispatch{},
                                                   dict_col.indices(),
                                                   *index,
                                                   stream)
                           : false;
}

}  // namespace

namespace detail {
bool contains(column_view const& col, scalar const& value, rmm::cuda_stream_view stream)
{
  if (col.is_empty()) { return false; }

  if (not value.is_valid()) { return col.has_nulls(); }

  return cudf::type_dispatcher(col.type(), contains_scalar_dispatch{}, col, value, stream);
}

struct multi_contains_dispatch {
  template <typename Element>
  std::unique_ptr<column> operator()(column_view const& haystack,
                                     column_view const& needles,
                                     rmm::cuda_stream_view stream,
                                     rmm::mr::device_memory_resource* mr)
  {
    std::unique_ptr<column> result = make_numeric_column(data_type{type_to_id<bool>()},
                                                         haystack.size(),
                                                         copy_bitmask(haystack),
                                                         haystack.null_count(),
                                                         stream,
                                                         mr);

    if (haystack.is_empty()) { return result; }

    mutable_column_view result_view = result.get()->mutable_view();

    if (needles.is_empty()) {
      thrust::fill(rmm::exec_policy(stream)->on(stream.value()),
                   result_view.begin<bool>(),
                   result_view.end<bool>(),
                   false);
      return result;
    }

    auto hash_set = cudf::detail::unordered_multiset<Element>::create(needles, stream.value());
    auto device_hash_set = hash_set.to_device();

    auto d_haystack_ptr = column_device_view::create(haystack, stream);
    auto d_haystack     = *d_haystack_ptr;

    if (haystack.has_nulls()) {
      thrust::transform(rmm::exec_policy(stream)->on(stream.value()),
                        thrust::make_counting_iterator<size_type>(0),
                        thrust::make_counting_iterator<size_type>(haystack.size()),
                        result_view.begin<bool>(),
                        [device_hash_set, d_haystack] __device__(size_t index) {
                          return d_haystack.is_null_nocheck(index) ||
                                 device_hash_set.contains(d_haystack.element<Element>(index));
                        });
    } else {
      thrust::transform(rmm::exec_policy(stream)->on(stream.value()),
                        thrust::make_counting_iterator<size_type>(0),
                        thrust::make_counting_iterator<size_type>(haystack.size()),
                        result_view.begin<bool>(),
                        [device_hash_set, d_haystack] __device__(size_t index) {
                          return device_hash_set.contains(d_haystack.element<Element>(index));
                        });
    }

    return result;
  }
};

template <>
std::unique_ptr<column> multi_contains_dispatch::operator()<list_view>(
  column_view const& haystack,
  column_view const& needles,
  rmm::cuda_stream_view stream,
  rmm::mr::device_memory_resource* mr)
{
  CUDF_FAIL("list_view type not supported");
}

template <>
std::unique_ptr<column> multi_contains_dispatch::operator()<struct_view>(
  column_view const& haystack,
  column_view const& needles,
  rmm::cuda_stream_view stream,
  rmm::mr::device_memory_resource* mr)
{
  CUDF_FAIL("struct_view type not supported");
}

template <>
std::unique_ptr<column> multi_contains_dispatch::operator()<dictionary32>(
  column_view const& haystack_in,
  column_view const& needles_in,
  rmm::cuda_stream_view stream,
  rmm::mr::device_memory_resource* mr)
{
  dictionary_column_view const haystack(haystack_in);
  dictionary_column_view const needles(needles_in);
  // first combine keys so both dictionaries have the same set
  auto haystack_matched    = dictionary::detail::add_keys(haystack, needles.keys(), stream);
  auto const haystack_view = dictionary_column_view(haystack_matched->view());
  auto needles_matched     = dictionary::detail::set_keys(needles, haystack_view.keys(), stream);
  auto const needles_view  = dictionary_column_view(needles_matched->view());

  // now just use the indices for the contains
  column_view const haystack_indices = haystack_view.get_indices_annotated();
  column_view const needles_indices  = needles_view.get_indices_annotated();
  return cudf::type_dispatcher(haystack_indices.type(),
                               multi_contains_dispatch{},
                               haystack_indices,
                               needles_indices,
                               stream,
                               mr);
}

std::unique_ptr<column> contains(column_view const& haystack,
                                 column_view const& needles,
                                 rmm::cuda_stream_view stream,
                                 rmm::mr::device_memory_resource* mr)
{
  CUDF_EXPECTS(haystack.type() == needles.type(), "DTYPE mismatch");

  return cudf::type_dispatcher(
    haystack.type(), multi_contains_dispatch{}, haystack, needles, stream, mr);
}

std::unique_ptr<column> lower_bound(table_view const& t,
                                    table_view const& values,
                                    std::vector<order> const& column_order,
                                    std::vector<null_order> const& null_precedence,
                                    rmm::cuda_stream_view stream,
                                    rmm::mr::device_memory_resource* mr)
{
  return search_ordered(t, values, true, column_order, null_precedence, stream, mr);
}

std::unique_ptr<column> upper_bound(table_view const& t,
                                    table_view const& values,
                                    std::vector<order> const& column_order,
                                    std::vector<null_order> const& null_precedence,
                                    rmm::cuda_stream_view stream,
                                    rmm::mr::device_memory_resource* mr)
{
  return search_ordered(t, values, false, column_order, null_precedence, stream, mr);
}

}  // namespace detail

// external APIs

std::unique_ptr<column> lower_bound(table_view const& t,
                                    table_view const& values,
                                    std::vector<order> const& column_order,
                                    std::vector<null_order> const& null_precedence,
                                    rmm::mr::device_memory_resource* mr)
{
  CUDF_FUNC_RANGE();
  return detail::lower_bound(
    t, values, column_order, null_precedence, rmm::cuda_stream_default, mr);
}

std::unique_ptr<column> upper_bound(table_view const& t,
                                    table_view const& values,
                                    std::vector<order> const& column_order,
                                    std::vector<null_order> const& null_precedence,
                                    rmm::mr::device_memory_resource* mr)
{
  CUDF_FUNC_RANGE();
  return detail::upper_bound(
    t, values, column_order, null_precedence, rmm::cuda_stream_default, mr);
}

bool contains(column_view const& col, scalar const& value)
{
  CUDF_FUNC_RANGE();
  return detail::contains(col, value, rmm::cuda_stream_default);
}

std::unique_ptr<column> contains(column_view const& haystack,
                                 column_view const& needles,
                                 rmm::mr::device_memory_resource* mr)
{
  CUDF_FUNC_RANGE();
  return detail::contains(haystack, needles, rmm::cuda_stream_default, mr);
}

}  // namespace cudf
