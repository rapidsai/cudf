#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <cudf/detail/column_factories.hpp>
#include <cudf/column/column_factories.hpp>
#include <cudf/search.hpp>
#include <cudf/legacy/copying.hpp>
#include <cudf/table/row_operators.cuh>
#include <cudf/table/table_device_view.cuh>
#include <cudf/table/table_view.hpp>
#include <cudf/scalar/scalar_device_view.cuh>
#include <hash/unordered_multiset.cuh>

#include <rmm/thrust_rmm_allocator.h>
#include <strings/utilities.hpp>

#include <thrust/binary_search.h>
#include <thrust/device_vector.h>
#include <thrust/logical.h>

namespace cudf {
namespace experimental {

namespace {

template <typename DataIterator, typename ValuesIterator,
          typename OutputIterator, typename Comparator>
void launch_search(DataIterator it_data,
                   ValuesIterator it_vals,
                   size_type data_size,
                   size_type values_size,
                   OutputIterator it_output,
                   Comparator comp,
                   bool find_first,
                   hipStream_t stream)
{
  if (find_first) {
    thrust::lower_bound(rmm::exec_policy(stream)->on(stream),
                        it_data, it_data + data_size,
                        it_vals, it_vals + values_size,
                        it_output, comp);
  }
  else {
    thrust::upper_bound(rmm::exec_policy(stream)->on(stream),
                        it_data, it_data + data_size,
                        it_vals, it_vals + values_size,
                        it_output, comp);
  }
}

} // namespace

namespace detail {

std::unique_ptr<column> search_ordered(table_view const& t,
                                       table_view const& values,
                                       bool find_first,
                                       std::vector<order> const& column_order,
                                       std::vector<null_order> const& null_precedence,
                                       rmm::mr::device_memory_resource *mr,
                                       hipStream_t stream = 0)
{
  // Allocate result column
  std::unique_ptr<column> result = make_numeric_column(data_type{experimental::type_to_id<size_type>()}, values.num_rows(),
                                                       mask_state::UNALLOCATED, stream, mr);

  mutable_column_view result_view = result.get()->mutable_view();

  // Handle empty inputs
  if (t.num_rows() == 0) {
    CUDA_TRY(hipMemset(result_view.data<size_type>(), 0, values.num_rows() * sizeof(size_type)));
    return result;
  }

  if (not column_order.empty()) {
    CUDF_EXPECTS(
        static_cast<std::size_t>(t.num_columns()) == column_order.size(),
        "Mismatch between number of columns and column order.");
  }

  if (not null_precedence.empty()) {
    CUDF_EXPECTS(
        static_cast<std::size_t>(t.num_columns()) == null_precedence.size(),
        "Mismatch between number of columns and null precedence.");
  }

  auto d_t      = table_device_view::create(t, stream);
  auto d_values = table_device_view::create(values, stream);
  auto count_it = thrust::make_counting_iterator<size_type>(0);

  rmm::device_vector<order> d_column_order(column_order.begin(), column_order.end());
  rmm::device_vector<null_order> d_null_precedence(null_precedence.begin(), null_precedence.end());

  if (has_nulls(t)) {
    auto ineq_op = (find_first)
      ? row_lexicographic_comparator<true>(*d_t, *d_values, d_column_order.data().get(), d_null_precedence.data().get())
      : row_lexicographic_comparator<true>(*d_values, *d_t, d_column_order.data().get(), d_null_precedence.data().get());

    launch_search(count_it, count_it, t.num_rows(), values.num_rows(),
                  result_view.data<size_type>(), ineq_op, find_first, stream);
  } else {
    auto ineq_op = (find_first)
      ? row_lexicographic_comparator<false>(*d_t, *d_values, d_column_order.data().get(), d_null_precedence.data().get())
      : row_lexicographic_comparator<false>(*d_values, *d_t, d_column_order.data().get(), d_null_precedence.data().get());

    launch_search(count_it, count_it, t.num_rows(), values.num_rows(),
                  result_view.data<size_type>(), ineq_op, find_first, stream);
  }

  return result;
}

template <typename Element, bool nullable = true>
struct compare_with_value{
  compare_with_value(column_device_view c, Element val, bool val_is_valid, bool nulls_are_equal)

    : col{c}, value{val}, val_is_valid{val_is_valid}, nulls_are_equal{nulls_are_equal} {}

  __device__ bool operator()(size_type i) noexcept {
    if (nullable) {
      bool const col_is_null{col.nullable() and col.is_null(i)};
      if (col_is_null and not val_is_valid)
        return nulls_are_equal;
      else if (col_is_null == val_is_valid)
        return false;
    }
    
    return equality_compare<Element>(col.element<Element>(i), value);
  }

  column_device_view        col;
  Element                   value;
  bool val_is_valid;
  bool nulls_are_equal;
};

template <typename Element>
void populate_element(scalar const& value, Element &e) {
  using ScalarType = cudf::experimental::scalar_type_t<Element>;
  auto s1 = static_cast<const ScalarType *>(&value);

  e = s1->value();
}

template <>
void populate_element<string_view>(scalar const& value, string_view &e) {
  using ScalarType = cudf::experimental::scalar_type_t<string_view>;
  auto s1 = static_cast<const ScalarType *>(&value);

  e = string_view{s1->data(), s1->size()};
}
  
struct contains_scalar_dispatch {
  template <typename Element>
  bool operator()(column_view const& col, scalar const& value,
                  hipStream_t stream,
                  rmm::mr::device_memory_resource *mr) {

    auto d_col = column_device_view::create(col, stream);
    auto data_it = thrust::make_counting_iterator<size_type>(0);

    bool    element_is_valid{value.is_valid()};
    Element element;

    populate_element(value, element);

    if (col.has_nulls()) {
      auto eq_op = compare_with_value<Element, true>(*d_col, element, element_is_valid, true);

      return thrust::any_of(rmm::exec_policy(stream)->on(stream),
                            data_it, data_it + col.size(),
                            eq_op);
    } else {
      auto eq_op = compare_with_value<Element, false>(*d_col, element, element_is_valid, true);

      return thrust::any_of(rmm::exec_policy(stream)->on(stream),
                            data_it, data_it + col.size(),
                            eq_op);
    }
  }
};

bool contains(column_view const& col,
              scalar const& value,
              rmm::mr::device_memory_resource *mr,
              hipStream_t stream = 0)
{
  CUDF_EXPECTS(col.type() == value.type(), "DTYPE mismatch");

  if (col.size() == 0) {
    return false;
  }

  if (not value.is_valid()) {
    return col.has_nulls();
  }

  return cudf::experimental::type_dispatcher(col.type(),
                                             contains_scalar_dispatch{},
                                             col, value,
                                             stream, mr);
}

struct multi_contains_dispatch {
  template <typename Element>
  std::unique_ptr<column> operator()(column_view const& haystack,
                                     column_view const& needles,
                                     rmm::mr::device_memory_resource *mr,
                                     hipStream_t stream) {

    std::unique_ptr<column> result = detail::make_numeric_column(data_type{experimental::type_to_id<bool8>()},
                                                         haystack.size(),
                                                         copy_bitmask(haystack),
                                                         haystack.null_count(),
                                                         stream, mr);
  
    if (haystack.size() == 0) {
      return result;
    }

    mutable_column_view result_view = result.get()->mutable_view();

    if (needles.size() == 0) {
      bool8 f = false;
      thrust::fill(rmm::exec_policy(stream)->on(stream), result_view.begin<bool8>(), result_view.end<bool8>(), f);
      return result;
    }

    auto hash_set = cudf::detail::unordered_multiset<Element>::create(needles, stream);
    auto device_hash_set = hash_set.to_device();

    auto d_haystack_ptr = column_device_view::create(haystack, stream);
    auto d_haystack = *d_haystack_ptr;

    if (haystack.has_nulls()) {
      thrust::transform(rmm::exec_policy(stream)->on(stream),
                        thrust::make_counting_iterator<size_type>(0),
                        thrust::make_counting_iterator<size_type>(haystack.size()),
                        result_view.begin<bool8>(),
                        [device_hash_set, d_haystack] __device__ (size_t index) {
                          return d_haystack.is_null_nocheck(index) || device_hash_set.contains(d_haystack.element<Element>(index));
                        });
    } else {
      thrust::transform(rmm::exec_policy(stream)->on(stream),
                        thrust::make_counting_iterator<size_type>(0),
                        thrust::make_counting_iterator<size_type>(haystack.size()),
                        result_view.begin<bool8>(),
                        [device_hash_set, d_haystack] __device__ (size_t index) {
                          return device_hash_set.contains(d_haystack.element<Element>(index));
                        });
    }

    return result;
  }
};

std::unique_ptr<column> contains(column_view const& haystack,
                                 column_view const& needles,
                                 rmm::mr::device_memory_resource* mr,
                                 hipStream_t stream = 0) {

  CUDF_EXPECTS(haystack.type() == needles.type(), "DTYPE mismatch");

  return cudf::experimental::type_dispatcher(haystack.type(),
                                             multi_contains_dispatch{},
                                             haystack, needles, mr, stream);
}
} // namespace detail

std::unique_ptr<column> lower_bound(table_view const& t,
                                    table_view const& values,
                                    std::vector<order> const& column_order,
                                    std::vector<null_order> const& null_precedence,
                                    rmm::mr::device_memory_resource *mr)
{
  return detail::search_ordered(t, values, true, column_order, null_precedence, mr);
}

std::unique_ptr<column> upper_bound(table_view const& t,
                                    table_view const& values,
                                    std::vector<order> const& column_order,
                                    std::vector<null_order> const& null_precedence,
                                    rmm::mr::device_memory_resource *mr)
{
  return detail::search_ordered(t, values, false, column_order, null_precedence, mr);
}

bool contains(column_view const& col, scalar const& value, rmm::mr::device_memory_resource *mr)
{
  return detail::contains(col, value, mr);
}

std::unique_ptr<column> contains(column_view const& haystack, column_view const& needles,
                                       rmm::mr::device_memory_resource* mr) {
  return detail::contains(haystack, needles, mr);
}

} // namespace exp
} // namespace cudf
