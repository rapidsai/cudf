#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2020, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <hipcub/hipcub.hpp>
#include <cudf/column/column_factories.hpp>
#include <cudf/copying.hpp>
#include <cudf/detail/gather.cuh>
#include <cudf/detail/nvtx/ranges.hpp>
#include <cudf/detail/scatter.cuh>
#include <cudf/detail/utilities/cuda.cuh>
#include <cudf/detail/utilities/hash_functions.cuh>
#include <cudf/partitioning.hpp>
#include <cudf/table/row_operators.cuh>
#include <cudf/table/table_device_view.cuh>

#include <rmm/cuda_stream_view.hpp>
#include <rmm/exec_policy.hpp>

namespace cudf {
namespace {
// Launch configuration for optimized hash partition
constexpr size_type OPTIMIZED_BLOCK_SIZE                     = 512;
constexpr size_type OPTIMIZED_ROWS_PER_THREAD                = 8;
constexpr size_type ELEMENTS_PER_THREAD                      = 2;
constexpr size_type THRESHOLD_FOR_OPTIMIZED_PARTITION_KERNEL = 1024;

// Launch configuration for fallback hash partition
constexpr size_type FALLBACK_BLOCK_SIZE      = 256;
constexpr size_type FALLBACK_ROWS_PER_THREAD = 1;

/**
 * @brief  Functor to map a hash value to a particular 'bin' or partition number
 * that uses the modulo operation.
 */
template <typename hash_value_t>
class modulo_partitioner {
 public:
  modulo_partitioner(size_type num_partitions) : divisor{num_partitions} {}

  __device__ size_type operator()(hash_value_t hash_value) const { return hash_value % divisor; }

 private:
  const size_type divisor;
};

template <typename T>
bool is_power_two(T number)
{
  return (0 == (number & (number - 1)));
}

/**
 * @brief  Functor to map a hash value to a particular 'bin' or partition number
 * that uses a bitwise mask. Only works when num_partitions is a power of 2.
 *
 * For n % d, if d is a power of two, then it can be computed more efficiently
 * via a single bitwise AND as: n & (d - 1)
 */
template <typename hash_value_t>
class bitwise_partitioner {
 public:
  bitwise_partitioner(size_type num_partitions) : mask{(num_partitions - 1)}
  {
    assert(is_power_two(num_partitions));
  }

  __device__ size_type operator()(hash_value_t hash_value) const
  {
    return hash_value & mask;  // hash_value & (num_partitions - 1)
  }

 private:
  const size_type mask;
};

/**
 * @brief Computes which partition each row of a device_table will belong to
 based on hashing each row, and applying a partition function to the hash value.
   Records the size of each partition for each thread block as well as the
 global size of each partition across all thread blocks.
 *
 * @param[in] the_table The table whose rows will be partitioned
 * @param[in] num_rows The number of rows in the table
 * @param[in] num_partitions The number of partitions to divide the rows into
 * @param[in] the_partitioner The functor that maps a rows hash value to a
 partition number
 * @param[out] row_partition_numbers Array that holds which partition each row
 belongs to
 * @param[out] row_partition_offset Array that holds the offset of each row in
 its partition of
 * the thread block
 * @param[out] block_partition_sizes Array that holds the size of each partition
 for each block,
 * i.e., { {block0 partition0 size, block1 partition0 size, ...},
         {block0 partition1 size, block1 partition1 size, ...},
         ...
         {block0 partition(num_partitions-1) size, block1
 partition(num_partitions -1) size, ...} }
 * @param[out] global_partition_sizes The number of rows in each partition.
 */
template <class row_hasher_t, typename partitioner_type>
__global__ void compute_row_partition_numbers(row_hasher_t the_hasher,
                                              const size_type num_rows,
                                              const size_type num_partitions,
                                              const partitioner_type the_partitioner,
                                              size_type* __restrict__ row_partition_numbers,
                                              size_type* __restrict__ row_partition_offset,
                                              size_type* __restrict__ block_partition_sizes,
                                              size_type* __restrict__ global_partition_sizes)
{
  // Accumulate histogram of the size of each partition in shared memory
  extern __shared__ size_type shared_partition_sizes[];

  size_type row_number = threadIdx.x + blockIdx.x * blockDim.x;

  // Initialize local histogram
  size_type partition_number = threadIdx.x;
  while (partition_number < num_partitions) {
    shared_partition_sizes[partition_number] = 0;
    partition_number += blockDim.x;
  }

  __syncthreads();

  // Compute the hash value for each row, store it to the array of hash values
  // and compute the partition to which the hash value belongs and increment
  // the shared memory counter for that partition
  while (row_number < num_rows) {
    const hash_value_type row_hash_value = the_hasher(row_number);

    const size_type partition_number = the_partitioner(row_hash_value);

    row_partition_numbers[row_number] = partition_number;

    row_partition_offset[row_number] =
      atomicAdd(&(shared_partition_sizes[partition_number]), size_type(1));

    row_number += blockDim.x * gridDim.x;
  }

  __syncthreads();

  // Flush shared memory histogram to global memory
  partition_number = threadIdx.x;
  while (partition_number < num_partitions) {
    const size_type block_partition_size = shared_partition_sizes[partition_number];

    // Update global size of each partition
    atomicAdd(&global_partition_sizes[partition_number], block_partition_size);

    // Record the size of this partition in this block
    const size_type write_location        = partition_number * gridDim.x + blockIdx.x;
    block_partition_sizes[write_location] = block_partition_size;
    partition_number += blockDim.x;
  }
}

/**
 * @brief  Given an array of partition numbers, computes the final output
 location for each element in the output such that all rows with the same
 partition are contiguous in memory.
 *
 * @param row_partition_numbers The array that records the partition number for
 each row
 * @param num_rows The number of rows
 * @param num_partitions THe number of partitions
 * @param[out] block_partition_offsets Array that holds the offset of each
 partition for each thread block,
 * i.e., { {block0 partition0 offset, block1 partition0 offset, ...},
         {block0 partition1 offset, block1 partition1 offset, ...},
         ...
         {block0 partition(num_partitions-1) offset, block1
 partition(num_partitions -1) offset, ...} }
 */
__global__ void compute_row_output_locations(size_type* __restrict__ row_partition_numbers,
                                             const size_type num_rows,
                                             const size_type num_partitions,
                                             size_type* __restrict__ block_partition_offsets)
{
  // Shared array that holds the offset of this blocks partitions in
  // global memory
  extern __shared__ size_type shared_partition_offsets[];

  // Initialize array of this blocks offsets from global array
  size_type partition_number = threadIdx.x;
  while (partition_number < num_partitions) {
    shared_partition_offsets[partition_number] =
      block_partition_offsets[partition_number * gridDim.x + blockIdx.x];
    partition_number += blockDim.x;
  }
  __syncthreads();

  size_type row_number = threadIdx.x + blockIdx.x * blockDim.x;

  // Get each row's partition number, and get it's output location by
  // incrementing block's offset counter for that partition number
  // and store the row's output location in-place
  while (row_number < num_rows) {
    // Get partition number of this row
    const size_type partition_number = row_partition_numbers[row_number];

    // Get output location based on partition number by incrementing the
    // corresponding partition offset for this block
    const size_type row_output_location =
      atomicAdd(&(shared_partition_offsets[partition_number]), size_type(1));

    // Store the row's output location in-place
    row_partition_numbers[row_number] = row_output_location;

    row_number += blockDim.x * gridDim.x;
  }
}

/**
 * @brief Move one column from the input table to the hashed table.
 *
 * @param[in] input_buf Data buffer of the column in the input table
 * @param[out] output_buf Preallocated data buffer of the column in the output
 * table
 * @param[in] num_rows The number of rows in each column
 * @param[in] num_partitions The number of partitions to divide the rows into
 * @param[in] row_partition_numbers Array that holds which partition each row
 * belongs to
 * @param[in] row_partition_offset Array that holds the offset of each row in
 * its partition of the thread block.
 * @param[in] block_partition_sizes Array that holds the size of each partition
 * for each block
 * @param[in] scanned_block_partition_sizes The scan of block_partition_sizes
 */
template <typename InputIter, typename DataType>
__global__ void copy_block_partitions(InputIter input_iter,
                                      DataType* __restrict__ output_buf,
                                      const size_type num_rows,
                                      const size_type num_partitions,
                                      size_type const* __restrict__ row_partition_numbers,
                                      size_type const* __restrict__ row_partition_offset,
                                      size_type const* __restrict__ block_partition_sizes,
                                      size_type const* __restrict__ scanned_block_partition_sizes)
{
  extern __shared__ char shared_memory[];
  auto block_output = reinterpret_cast<DataType*>(shared_memory);
  auto partition_offset_shared =
    reinterpret_cast<size_type*>(block_output + OPTIMIZED_BLOCK_SIZE * OPTIMIZED_ROWS_PER_THREAD);
  auto partition_offset_global = partition_offset_shared + num_partitions + 1;

  typedef hipcub::BlockScan<size_type, OPTIMIZED_BLOCK_SIZE> BlockScan;
  __shared__ typename BlockScan::TempStorage temp_storage;

  // use ELEMENTS_PER_THREAD=2 to support upto 1024 partitions
  size_type temp_histo[ELEMENTS_PER_THREAD];

  for (int i = 0; i < ELEMENTS_PER_THREAD; ++i) {
    if (ELEMENTS_PER_THREAD * threadIdx.x + i < num_partitions) {
      temp_histo[i] =
        block_partition_sizes[blockIdx.x + (ELEMENTS_PER_THREAD * threadIdx.x + i) * gridDim.x];
    } else {
      temp_histo[i] = 0;
    }
  }

  __syncthreads();

  BlockScan(temp_storage).InclusiveSum(temp_histo, temp_histo);

  __syncthreads();

  if (threadIdx.x == 0) { partition_offset_shared[0] = 0; }

  // Calculate the offset in shared memory of each partition in this thread
  // block
  for (int i = 0; i < ELEMENTS_PER_THREAD; ++i) {
    if (ELEMENTS_PER_THREAD * threadIdx.x + i < num_partitions) {
      partition_offset_shared[ELEMENTS_PER_THREAD * threadIdx.x + i + 1] = temp_histo[i];
    }
  }

  // Fetch the offset in the output buffer of each partition in this thread
  // block
  for (size_type ipartition = threadIdx.x; ipartition < num_partitions; ipartition += blockDim.x) {
    partition_offset_global[ipartition] =
      scanned_block_partition_sizes[ipartition * gridDim.x + blockIdx.x];
  }

  __syncthreads();

  // Fetch the input data to shared memory
  for (size_type row_number = threadIdx.x + blockIdx.x * blockDim.x; row_number < num_rows;
       row_number += blockDim.x * gridDim.x) {
    size_type const ipartition = row_partition_numbers[row_number];

    block_output[partition_offset_shared[ipartition] + row_partition_offset[row_number]] =
      input_iter[row_number];
  }

  __syncthreads();

  // Copy data from shared memory to output using 32 threads for each partition
  constexpr int nthreads_partition = 32;
  static_assert(OPTIMIZED_BLOCK_SIZE % nthreads_partition == 0,
                "BLOCK_SIZE must be divisible by number of threads");

  for (size_type ipartition = threadIdx.x / nthreads_partition; ipartition < num_partitions;
       ipartition += OPTIMIZED_BLOCK_SIZE / nthreads_partition) {
    size_type const nelements_partition =
      partition_offset_shared[ipartition + 1] - partition_offset_shared[ipartition];

    for (size_type row_offset = threadIdx.x % nthreads_partition; row_offset < nelements_partition;
         row_offset += nthreads_partition) {
      output_buf[partition_offset_global[ipartition] + row_offset] =
        block_output[partition_offset_shared[ipartition] + row_offset];
    }
  }
}

template <typename InputIter, typename OutputIter>
void copy_block_partitions_impl(InputIter const input,
                                OutputIter output,
                                size_type num_rows,
                                size_type num_partitions,
                                size_type const* row_partition_numbers,
                                size_type const* row_partition_offset,
                                size_type const* block_partition_sizes,
                                size_type const* scanned_block_partition_sizes,
                                size_type grid_size,
                                rmm::cuda_stream_view stream)
{
  // We need 3 chunks of shared memory:
  // 1. BLOCK_SIZE * ROWS_PER_THREAD elements of size_type for copying to output
  // 2. num_partitions + 1 elements of size_type for per-block partition offsets
  // 3. num_partitions + 1 elements of size_type for global partition offsets
  int const smem = OPTIMIZED_BLOCK_SIZE * OPTIMIZED_ROWS_PER_THREAD * sizeof(*output) +
                   (num_partitions + 1) * sizeof(size_type) * 2;

  copy_block_partitions<<<grid_size, OPTIMIZED_BLOCK_SIZE, smem, stream.value()>>>(
    input,
    output,
    num_rows,
    num_partitions,
    row_partition_numbers,
    row_partition_offset,
    block_partition_sizes,
    scanned_block_partition_sizes);
}

rmm::device_vector<size_type> compute_gather_map(size_type num_rows,
                                                 size_type num_partitions,
                                                 size_type const* row_partition_numbers,
                                                 size_type const* row_partition_offset,
                                                 size_type const* block_partition_sizes,
                                                 size_type const* scanned_block_partition_sizes,
                                                 size_type grid_size,
                                                 rmm::cuda_stream_view stream)
{
  auto sequence = thrust::make_counting_iterator(0);
  rmm::device_vector<size_type> gather_map(num_rows);

  copy_block_partitions_impl(sequence,
                             gather_map.data().get(),
                             num_rows,
                             num_partitions,
                             row_partition_numbers,
                             row_partition_offset,
                             block_partition_sizes,
                             scanned_block_partition_sizes,
                             grid_size,
                             stream);

  return gather_map;
}

struct copy_block_partitions_dispatcher {
  template <typename DataType, std::enable_if_t<is_fixed_width<DataType>()>* = nullptr>
  std::unique_ptr<column> operator()(column_view const& input,
                                     const size_type num_partitions,
                                     size_type const* row_partition_numbers,
                                     size_type const* row_partition_offset,
                                     size_type const* block_partition_sizes,
                                     size_type const* scanned_block_partition_sizes,
                                     size_type grid_size,
                                     rmm::cuda_stream_view stream,
                                     rmm::mr::device_memory_resource* mr)
  {
    using Type = device_storage_type_t<DataType>;

    rmm::device_buffer output(input.size() * sizeof(Type), stream, mr);

    copy_block_partitions_impl(input.data<Type>(),
                               static_cast<Type*>(output.data()),
                               input.size(),
                               num_partitions,
                               row_partition_numbers,
                               row_partition_offset,
                               block_partition_sizes,
                               scanned_block_partition_sizes,
                               grid_size,
                               stream);

    return std::make_unique<column>(input.type(), input.size(), std::move(output));
  }

  template <typename DataType, std::enable_if_t<not is_fixed_width<DataType>()>* = nullptr>
  std::unique_ptr<column> operator()(column_view const& input,
                                     const size_type num_partitions,
                                     size_type const* row_partition_numbers,
                                     size_type const* row_partition_offset,
                                     size_type const* block_partition_sizes,
                                     size_type const* scanned_block_partition_sizes,
                                     size_type grid_size,
                                     rmm::cuda_stream_view stream,
                                     rmm::mr::device_memory_resource* mr)
  {
    // Use move_to_output_buffer to create an equivalent gather map
    auto gather_map = compute_gather_map(input.size(),
                                         num_partitions,
                                         row_partition_numbers,
                                         row_partition_offset,
                                         block_partition_sizes,
                                         scanned_block_partition_sizes,
                                         grid_size,
                                         stream);

    // Use gather instead for non-fixed width types
    return type_dispatcher(input.type(),
                           detail::column_gatherer{},
                           input,
                           gather_map.begin(),
                           gather_map.end(),
                           false,
                           stream,
                           mr);
  }
};

// NOTE hash_has_nulls must be true if table_to_hash has nulls
template <template <typename> class hash_function, bool hash_has_nulls>
std::pair<std::unique_ptr<table>, std::vector<size_type>> hash_partition_table(
  table_view const& input,
  table_view const& table_to_hash,
  size_type num_partitions,
  rmm::cuda_stream_view stream,
  rmm::mr::device_memory_resource* mr)
{
  auto const num_rows = table_to_hash.num_rows();

  bool const use_optimization{num_partitions <= THRESHOLD_FOR_OPTIMIZED_PARTITION_KERNEL};
  auto const block_size = use_optimization ? OPTIMIZED_BLOCK_SIZE : FALLBACK_BLOCK_SIZE;
  auto const rows_per_thread =
    use_optimization ? OPTIMIZED_ROWS_PER_THREAD : FALLBACK_ROWS_PER_THREAD;
  auto const rows_per_block = block_size * rows_per_thread;

  // NOTE grid_size is non-const to workaround lambda capture bug in gcc 5.4
  auto grid_size = util::div_rounding_up_safe(num_rows, rows_per_block);

  // Allocate array to hold which partition each row belongs to
  auto row_partition_numbers = rmm::device_vector<size_type>(num_rows);

  // Array to hold the size of each partition computed by each block
  //  i.e., { {block0 partition0 size, block1 partition0 size, ...},
  //          {block0 partition1 size, block1 partition1 size, ...},
  //          ...
  //          {block0 partition(num_partitions-1) size, block1
  //          partition(num_partitions -1) size, ...} }
  auto block_partition_sizes = rmm::device_vector<size_type>(grid_size * num_partitions);

  auto scanned_block_partition_sizes = rmm::device_vector<size_type>(grid_size * num_partitions);

  // Holds the total number of rows in each partition
  auto global_partition_sizes = rmm::device_vector<size_type>(num_partitions, size_type{0});

  auto row_partition_offset = rmm::device_vector<size_type>(num_rows);

  auto const device_input = table_device_view::create(table_to_hash, stream);
  auto const hasher       = row_hasher<hash_function, hash_has_nulls>(*device_input);

  // If the number of partitions is a power of two, we can compute the partition
  // number of each row more efficiently with bitwise operations
  if (is_power_two(num_partitions)) {
    // Determines how the mapping between hash value and partition number is
    // computed
    using partitioner_type = bitwise_partitioner<hash_value_type>;

    // Computes which partition each row belongs to by hashing the row and
    // performing a partitioning operator on the hash value. Also computes the
    // number of rows in each partition both for each thread block as well as
    // across all blocks
    compute_row_partition_numbers<<<grid_size,
                                    block_size,
                                    num_partitions * sizeof(size_type),
                                    stream.value()>>>(hasher,
                                                      num_rows,
                                                      num_partitions,
                                                      partitioner_type(num_partitions),
                                                      row_partition_numbers.data().get(),
                                                      row_partition_offset.data().get(),
                                                      block_partition_sizes.data().get(),
                                                      global_partition_sizes.data().get());
  } else {
    // Determines how the mapping between hash value and partition number is
    // computed
    using partitioner_type = modulo_partitioner<hash_value_type>;

    // Computes which partition each row belongs to by hashing the row and
    // performing a partitioning operator on the hash value. Also computes the
    // number of rows in each partition both for each thread block as well as
    // across all blocks
    compute_row_partition_numbers<<<grid_size,
                                    block_size,
                                    num_partitions * sizeof(size_type),
                                    stream.value()>>>(hasher,
                                                      num_rows,
                                                      num_partitions,
                                                      partitioner_type(num_partitions),
                                                      row_partition_numbers.data().get(),
                                                      row_partition_offset.data().get(),
                                                      block_partition_sizes.data().get(),
                                                      global_partition_sizes.data().get());
  }

  // Compute exclusive scan of all blocks' partition sizes in-place to determine
  // the starting point for each blocks portion of each partition in the output
  thrust::exclusive_scan(rmm::exec_policy(stream),
                         block_partition_sizes.begin(),
                         block_partition_sizes.end(),
                         scanned_block_partition_sizes.data().get());

  // Compute exclusive scan of size of each partition to determine offset
  // location of each partition in final output.
  // TODO This can be done independently on a separate stream
  size_type* scanned_global_partition_sizes{global_partition_sizes.data().get()};
  thrust::exclusive_scan(rmm::exec_policy(stream),
                         global_partition_sizes.begin(),
                         global_partition_sizes.end(),
                         scanned_global_partition_sizes);

  // Copy the result of the exclusive scan to the output offsets array
  // to indicate the starting point for each partition in the output
  std::vector<size_type> partition_offsets(num_partitions);
  CUDA_TRY(hipMemcpyAsync(partition_offsets.data(),
                           scanned_global_partition_sizes,
                           num_partitions * sizeof(size_type),
                           hipMemcpyDeviceToHost,
                           stream.value()));

  // When the number of partitions is less than a threshold, we can apply an
  // optimization using shared memory to copy values to the output buffer.
  // Otherwise, fallback to using scatter.
  if (use_optimization) {
    std::vector<std::unique_ptr<column>> output_cols(input.num_columns());

    // NOTE these pointers are non-const to workaround lambda capture bug in
    // gcc 5.4
    auto row_partition_numbers_ptr{row_partition_numbers.data().get()};
    auto row_partition_offset_ptr{row_partition_offset.data().get()};
    auto block_partition_sizes_ptr{block_partition_sizes.data().get()};
    auto scanned_block_partition_sizes_ptr{scanned_block_partition_sizes.data().get()};

    // Copy input to output by partition per column
    std::transform(input.begin(), input.end(), output_cols.begin(), [=](auto const& col) {
      return cudf::type_dispatcher(col.type(),
                                   copy_block_partitions_dispatcher{},
                                   col,
                                   num_partitions,
                                   row_partition_numbers_ptr,
                                   row_partition_offset_ptr,
                                   block_partition_sizes_ptr,
                                   scanned_block_partition_sizes_ptr,
                                   grid_size,
                                   stream,
                                   mr);
    });

    if (has_nulls(input)) {
      // Use copy_block_partitions to compute a gather map
      auto gather_map = compute_gather_map(num_rows,
                                           num_partitions,
                                           row_partition_numbers_ptr,
                                           row_partition_offset_ptr,
                                           block_partition_sizes_ptr,
                                           scanned_block_partition_sizes_ptr,
                                           grid_size,
                                           stream);

      // Handle bitmask using gather to take advantage of ballot_sync
      detail::gather_bitmask(
        input, gather_map.begin(), output_cols, detail::gather_bitmask_op::DONT_CHECK, stream, mr);
    }

    auto output{std::make_unique<table>(std::move(output_cols))};
    return std::make_pair(std::move(output), std::move(partition_offsets));
  } else {
    // Compute a scatter map from input to output such that the output rows are
    // sorted by partition number
    auto row_output_locations{row_partition_numbers.data().get()};
    auto scanned_block_partition_sizes_ptr{scanned_block_partition_sizes.data().get()};
    compute_row_output_locations<<<grid_size,
                                   block_size,
                                   num_partitions * sizeof(size_type),
                                   stream.value()>>>(
      row_output_locations, num_rows, num_partitions, scanned_block_partition_sizes_ptr);

    // Use the resulting scatter map to materialize the output
    auto output = detail::scatter(
      input, row_partition_numbers.begin(), row_partition_numbers.end(), input, false, stream, mr);

    return std::make_pair(std::move(output), std::move(partition_offsets));
  }
}

struct dispatch_map_type {
  /**
   * @brief Partitions the table `t` according to the `partition_map`.
   *
   * Algorithm:
   * - Compute the histogram of the size each partition
   * - Compute the exclusive scan of the histogram to get the offset for each
   * partition in the final partitioned output
   * - Use a transform iterator to materialize the scatter map of the rows from
   * `t` into the final output.
   *
   * @note JH: It would likely be more efficient to avoid the atomic increments
   * in the transform iterator. It would probably be faster to compute a
   * per-thread block histogram and compute an exclusive scan of all of the
   * per-block histograms (like in hash partition). But I'm purposefully trying
   * to reduce memory pressure by avoiding intermediate materializations. Plus,
   * atomics resolve in L2 and should be pretty fast since all the offsets will
   * fit in L2.
   *
   */
  template <typename MapType>
  std::enable_if_t<is_index_type<MapType>(),
                   std::pair<std::unique_ptr<table>, std::vector<size_type>>>
  operator()(table_view const& t,
             column_view const& partition_map,
             size_type num_partitions,
             rmm::cuda_stream_view stream,
             rmm::mr::device_memory_resource* mr) const
  {
    // Build a histogram of the number of rows in each partition
    rmm::device_vector<size_type> histogram(num_partitions + 1);
    std::size_t temp_storage_bytes{};
    std::size_t const num_levels = num_partitions + 1;
    size_type const lower_level  = 0;
    size_type const upper_level  = num_partitions;
    hipcub::DeviceHistogram::HistogramEven(nullptr,
                                        temp_storage_bytes,
                                        partition_map.begin<MapType>(),
                                        histogram.data().get(),
                                        num_levels,
                                        lower_level,
                                        upper_level,
                                        partition_map.size(),
                                        stream.value());

    rmm::device_buffer temp_storage(temp_storage_bytes, stream);

    hipcub::DeviceHistogram::HistogramEven(temp_storage.data(),
                                        temp_storage_bytes,
                                        partition_map.begin<MapType>(),
                                        histogram.data().get(),
                                        num_levels,
                                        lower_level,
                                        upper_level,
                                        partition_map.size(),
                                        stream.value());

    // `histogram` was created with an extra entry at the end such that an
    // exclusive scan will put the total number of rows at the end
    thrust::exclusive_scan(
      rmm::exec_policy(stream), histogram.begin(), histogram.end(), histogram.begin());

    // Copy offsets to host
    std::vector<size_type> partition_offsets(histogram.size());
    thrust::copy(histogram.begin(), histogram.end(), partition_offsets.begin());

    // Unfortunately need to materialize the scatter map because
    // `detail::scatter` requires multiple passes through the iterator
    rmm::device_vector<MapType> scatter_map(partition_map.size());

    // For each `partition_map[i]`, atomically increment the corresponding
    // partition offset to determine `i`s location in the output
    thrust::transform(rmm::exec_policy(stream),
                      partition_map.begin<MapType>(),
                      partition_map.end<MapType>(),
                      scatter_map.begin(),
                      [offsets = histogram.data().get()] __device__(auto partition_number) {
                        return atomicAdd(&offsets[partition_number], 1);
                      });

    // Scatter the rows into their partitions
    auto scattered =
      cudf::detail::scatter(t, scatter_map.begin(), scatter_map.end(), t, false, stream, mr);

    return std::make_pair(std::move(scattered), std::move(partition_offsets));
  }

  template <typename MapType>
  std::enable_if_t<not is_index_type<MapType>(),
                   std::pair<std::unique_ptr<table>, std::vector<size_type>>>
  operator()(table_view const& t,
             column_view const& partition_map,
             size_type num_partitions,
             rmm::cuda_stream_view stream,
             rmm::mr::device_memory_resource* mr) const
  {
    CUDF_FAIL("Unexpected, non-integral partition map.");
  }
};
}  // namespace

namespace detail {
namespace local {
template <template <typename> class hash_function>
std::pair<std::unique_ptr<table>, std::vector<size_type>> hash_partition(
  table_view const& input,
  std::vector<size_type> const& columns_to_hash,
  int num_partitions,
  rmm::cuda_stream_view stream,
  rmm::mr::device_memory_resource* mr)
{
  auto table_to_hash = input.select(columns_to_hash);

  // Return empty result if there are no partitions or nothing to hash
  if (num_partitions <= 0 || input.num_rows() == 0 || table_to_hash.num_columns() == 0) {
    return std::make_pair(empty_like(input), std::vector<size_type>{});
  }

  if (has_nulls(table_to_hash)) {
    return hash_partition_table<hash_function, true>(
      input, table_to_hash, num_partitions, stream, mr);
  } else {
    return hash_partition_table<hash_function, false>(
      input, table_to_hash, num_partitions, stream, mr);
  }
}
}  // namespace local

std::pair<std::unique_ptr<table>, std::vector<size_type>> partition(
  table_view const& t,
  column_view const& partition_map,
  size_type num_partitions,
  rmm::cuda_stream_view stream,
  rmm::mr::device_memory_resource* mr)
{
  CUDF_EXPECTS(t.num_rows() == partition_map.size(),
               "Size mismatch between table and partition map.");
  CUDF_EXPECTS(not partition_map.has_nulls(), "Unexpected null values in partition_map.");

  if (num_partitions == 0 or t.num_rows() == 0) {
    return std::make_pair(empty_like(t), std::vector<size_type>{});
  }

  return cudf::type_dispatcher(
    partition_map.type(), dispatch_map_type{}, t, partition_map, num_partitions, stream, mr);
}
}  // namespace detail

// Partition based on hash values
std::pair<std::unique_ptr<table>, std::vector<size_type>> hash_partition(
  table_view const& input,
  std::vector<size_type> const& columns_to_hash,
  int num_partitions,
  hash_id hash_function,
  rmm::cuda_stream_view stream,
  rmm::mr::device_memory_resource* mr)
{
  CUDF_FUNC_RANGE();

  switch (hash_function) {
    case (hash_id::HASH_IDENTITY):
      for (const size_type& column_id : columns_to_hash) {
        if (!is_numeric(input.column(column_id).type()))
          CUDF_FAIL("IdentityHash does not support this data type");
      }
      return detail::local::hash_partition<IdentityHash>(
        input, columns_to_hash, num_partitions, stream, mr);
    case (hash_id::HASH_MURMUR3):
      return detail::local::hash_partition<MurmurHash3_32>(
        input, columns_to_hash, num_partitions, stream, mr);
    default: CUDF_FAIL("Unsupported hash function in hash_partition");
  }
}

// Partition based on an explicit partition map
std::pair<std::unique_ptr<table>, std::vector<size_type>> partition(
  table_view const& t,
  column_view const& partition_map,
  size_type num_partitions,
  rmm::mr::device_memory_resource* mr)
{
  CUDF_FUNC_RANGE();
  return detail::partition(t, partition_map, num_partitions, rmm::cuda_stream_default, mr);
}

}  // namespace cudf
