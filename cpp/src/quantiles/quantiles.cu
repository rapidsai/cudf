#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2018, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

//Quantile (percentile) functionality

#include <thrust/device_vector.h>
#include <thrust/copy.h>

#include "cudf.h"
#include "utilities/cudf_utils.h"
#include "utilities/error_utils.h"
#include "rmm/thrust_rmm_allocator.h"

#include "quantiles.h"

namespace{ //unknown
  template<typename VType,
           typename RetT = double>
    void f_quantile_tester(rmm::device_vector<VType>& d_in)
  {
    using FctrType = std::function<RetT(VType, VType, double)>;

    FctrType lin_interp{[](VType y0, VType y1, double x){
        return static_cast<RetT>(static_cast<double>(y0) + x*static_cast<double>(y1-y0));//(f(x) - y0) / (x - 0) = m = (y1 - y0)/(1 - 0)
      }};

    FctrType midpoint{[](VType y0, VType y1, double x){
        return static_cast<RetT>(static_cast<double>(y0 + y1)/2.0);
      }};

    FctrType nearest{[](VType y0, VType y1, double x){
        return static_cast<RetT>(x < 0.5 ? y0 : y1);
      }};

    FctrType lowest{[](VType y0, VType y1, double x){
        return static_cast<RetT>(y0);
      }};

    FctrType highest{[](VType y0, VType y1, double x){
        return static_cast<RetT>(y1);
      }};
  
  
    std::vector<std::string> methods{"lin_interp", "midpoint", "nearest", "lowest", "highest"};
    size_t n_methods = methods.size();
    std::vector<FctrType> vf{lin_interp, midpoint, nearest, lowest, highest};
  
    std::vector<double> qvals{0.0, 0.25, 0.33, 0.5, 1.0};

  
    assert( n_methods == methods.size() );
  
    for(auto q: qvals)
      {
        VType res = quantile_approx(d_in.data().get(), d_in.size(), q);
        std::cout<<"q: "<<q<<"; exact res: "<<res<<"\n";
        for(auto i = 0;i<n_methods;++i)
          {
            RetT rt = quantile_exact(d_in.data().get(), d_in.size(), q, vf[i]);
            std::cout<<"q: "<<q<<"; method: "<<methods[i]<<"; rt: "<<rt<<"\n";
          }
      }
  }

  template<typename ColType,
           typename RetT = double> // just in case double won't be enough to hold result, in the future
  gdf_error trampoline_exact(gdf_column*  col_in,
                             gdf_quantile_method prec,
                             double q,
                             void* t_erased_res,
                             gdf_context* ctxt)
  {
    RetT* ptr_res = static_cast<RetT*>(t_erased_res);
    size_t n = col_in->size;
    ColType* p_dv = static_cast<ColType*>(col_in->data);
    if( ctxt->flag_sort_inplace || ctxt->flag_sorted)
      {
        return select_quantile(p_dv,
                               n,
                               q, 
                               prec,
                               *ptr_res,
                               ctxt->flag_sorted);
      }
    else
      {
        rmm::device_vector<ColType> dv(n);
        thrust::copy_n(thrust::device, /*TODO: stream*/p_dv, n, dv.begin());
        hipDeviceSynchronize();
        p_dv = dv.data().get();

        return select_quantile(p_dv,
                               n,
                               q, 
                               prec,
                               *ptr_res,
                               ctxt->flag_sorted);
      }
  }

  template<typename ColType>
  void trampoline_approx(gdf_column*  col_in,
                         double q,
                         void* t_erased_res,
                         gdf_context* ctxt)
  {
    ColType* ptr_res = static_cast<ColType*>(t_erased_res);
    size_t n = col_in->size;
    ColType* p_dv = static_cast<ColType*>(col_in->data);
    if( ctxt->flag_sort_inplace || ctxt->flag_sorted )
      {
        *ptr_res = quantile_approx(p_dv, n, q, NULL, ctxt->flag_sorted);
      }
    else
      {
        rmm::device_vector<ColType> dv(n);
        thrust::copy_n(thrust::device, /*TODO: stream*/p_dv, n, dv.begin());
        hipDeviceSynchronize();
        p_dv = dv.data().get();

        *ptr_res = quantile_approx(p_dv, n, q, NULL, ctxt->flag_sorted);
      }
  }
    
}//unknown namespace

gdf_error gdf_quantile_exact(	gdf_column*         col_in,       //input column;
                                gdf_quantile_method prec,         //precision: type of quantile method calculation
                                double              q,            //requested quantile in [0,1]
                                void*               t_erased_res, //result; for <exact> should probably be double*; it's void* because
                                                                  //(1) for uniformity of interface with <approx>;
                                                                  //(2) for possible types bigger than double, in the future;
                                gdf_context*        ctxt)         //context info
{
  GDF_REQUIRE(!col_in->valid || !col_in->null_count, GDF_VALIDITY_UNSUPPORTED);
  gdf_error ret = GDF_SUCCESS;
  assert( col_in->size > 0 );
  
  switch( col_in->dtype )
    {
    case GDF_INT8:
      {
        using ColType = int8_t;//char;
        ret = trampoline_exact<ColType>(col_in, prec, q, t_erased_res, ctxt);
        
        break;
      }
    case GDF_INT16:
      {
        using ColType = int16_t;//short;
        ret = trampoline_exact<ColType>(col_in, prec, q, t_erased_res, ctxt);
	  
        break;
        
      }
    case GDF_INT32:
      {
        using ColType = int32_t;//int;
        ret = trampoline_exact<ColType>(col_in, prec, q, t_erased_res, ctxt);
	  
        break;
        
      }
    case GDF_INT64:
      {
        using ColType = int64_t;//long;
        ret = trampoline_exact<ColType>(col_in, prec, q, t_erased_res, ctxt);
	  
        break;
        
      }
    case GDF_FLOAT32:
      {
        using ColType = float;
        ret = trampoline_exact<ColType>(col_in, prec, q, t_erased_res, ctxt);
	  
        break;
      }
    case GDF_FLOAT64:
      {
        using ColType = double;
        ret = trampoline_exact<ColType>(col_in, prec, q, t_erased_res, ctxt);
	  
        break;
      }

    default:
      assert( false );//type not handled, yet
    }

  return ret;
}

gdf_error gdf_quantile_aprrox(	gdf_column*  col_in,       //input column;
                                double       q,            //requested quantile in [0,1]
                                void*        t_erased_res, //type-erased result of same type as column;
                                gdf_context* ctxt)         //context info
{
  GDF_REQUIRE(!col_in->valid || !col_in->null_count, GDF_VALIDITY_UNSUPPORTED);
  gdf_error ret = GDF_SUCCESS;
  assert( col_in->size > 0 );
  
  switch( col_in->dtype )
    {
    case GDF_INT8:
      {
        using ColType = int8_t;//char;
        trampoline_approx<ColType>(col_in, q, t_erased_res, ctxt);
	  
        break;
      }
    case GDF_INT16:
      {
        using ColType = int16_t;//short;
        trampoline_approx<ColType>(col_in, q, t_erased_res, ctxt);
	  
        break;
        
      }
    case GDF_INT32:
      {
        using ColType = int32_t;//int;
        trampoline_approx<ColType>(col_in, q, t_erased_res, ctxt);
	  
        break;
        
      }
    case GDF_INT64:
      {
        using ColType = int64_t;//long;
        trampoline_approx<ColType>(col_in, q, t_erased_res, ctxt);
	  
        break;
        
      }
    case GDF_FLOAT32:
      {
        using ColType = float;
        trampoline_approx<ColType>(col_in, q, t_erased_res, ctxt);
	  
        break;
      }
    case GDF_FLOAT64:
      {
        using ColType = double;
        trampoline_approx<ColType>(col_in, q, t_erased_res, ctxt);
	  
        break;
      }

    default:
      assert( false );//type not handled, yet
    }

  return ret;
}

