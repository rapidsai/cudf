/*
 * Copyright (c) 2018, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

//Quantile (percentile) functionality

#include <cudf/cudf.h>
#include <quantiles/quantiles_util.hpp>
#include <utilities/cudf_utils.h>
#include <cudf/utilities/error.hpp>
#include <cudf/utilities/legacy/type_dispatcher.hpp>
#include <cudf/utilities/legacy/wrapper_types.hpp>
#include <rmm/thrust_rmm_allocator.h>

#include <thrust/device_vector.h>
#include <thrust/copy.h>


namespace{ // anonymous

  // compute quantile value as `result` from `quant` value by `interpolation` method
  template<typename T, typename RetT>
  gdf_error select_quantile(T* devarr,
                          cudf::size_type n,
                          double quant, 
                          cudf::interpolation interpolation,
                          RetT& result,
                          bool flag_sorted,
                          hipStream_t stream)
  {
    T hvalue;

    if( quant >= 1.0 && !flag_sorted )
    {
      T* d_res = thrust::max_element(rmm::exec_policy(stream)->on(stream), devarr, devarr+n);
      hvalue = cudf::detail::get_array_value(d_res, 0);
      result = static_cast<RetT>( hvalue );
      return GDF_SUCCESS;
    }

    if( quant <= 0.0 && !flag_sorted )
    {
      T* d_res = thrust::min_element(rmm::exec_policy(stream)->on(stream), devarr, devarr+n);
      hvalue = cudf::detail::get_array_value(d_res, 0);
      result = static_cast<RetT>( hvalue );
      return GDF_SUCCESS;
    }

    // sort if the input is not sorted.
    if( !flag_sorted ){
      thrust::sort(rmm::exec_policy(stream)->on(stream), devarr, devarr+n);
    }

    result = cudf::detail::select_quantile(devarr, n, quant, interpolation);
    
    return GDF_SUCCESS;
  }

  template<typename ColType,
           typename RetT = double>
  gdf_error trampoline_exact(gdf_column*  col_in,
                             cudf::interpolation interpolation,
                             double quant,
                             void* t_erased_res,
                             gdf_context* ctxt,
                             hipStream_t stream)
  {
    RetT* ptr_res = static_cast<RetT*>(t_erased_res);
    size_t n = col_in->size;
    ColType* col_data = static_cast<ColType*>(col_in->data);
    
    if( ctxt->flag_sort_inplace  && ctxt->flag_sorted )
    {
      return select_quantile(col_data,
                             n,
                             quant, 
                             interpolation,
                             *ptr_res,
                             ctxt->flag_sorted,
                             stream);
    }else{
      // create a clone of col_data if sort is required but sort_inplace is not allowed.
      rmm::device_vector<ColType> dv(n);
      thrust::copy_n(rmm::exec_policy(stream)->on(stream), col_data, n, dv.begin());
      ColType* clone_data = dv.data().get();

      return select_quantile(clone_data,
                             n,
                             quant, 
                             interpolation,
                             *ptr_res,
                             ctxt->flag_sorted,
                             stream);
    }
  }
    
  struct trampoline_exact_functor{
    template <typename T,
              typename std::enable_if_t<!std::is_arithmetic<T>::value, int> = 0>
    gdf_error operator()(gdf_column* col_in,
                         cudf::interpolation interpolation,
                         double              quant,
                         void*               t_erased_res,
                         gdf_context*        ctxt,
                         hipStream_t        stream = NULL)
    {
      return GDF_UNSUPPORTED_DTYPE;
    }

    template <typename T,
              typename std::enable_if_t<std::is_arithmetic<T>::value, int> = 0>
    gdf_error operator()(gdf_column*  col_in,
                         cudf::interpolation interpolation,
                         double              quant,
                         void*               t_erased_res,
                         gdf_context*        ctxt,
                         hipStream_t        stream = NULL)
    {
      // just in case double won't be enough to hold result
      // it can be changed in future
      return trampoline_exact<T, double>
                 (col_in, interpolation, quant, t_erased_res, ctxt, stream);
    }
  };

  struct trampoline_approx_functor{
    template <typename T,
              typename std::enable_if_t<!std::is_arithmetic<T>::value, int> = 0>
    gdf_error operator()(gdf_column* col_in,
                         double              quant,
                         void*               t_erased_res,
                         gdf_context*        ctxt,
                         hipStream_t        stream = NULL)
    {
      // TODO: support non-arithemetic types
      return GDF_UNSUPPORTED_DTYPE;
    }

    template <typename T,
              typename std::enable_if_t<std::is_arithmetic<T>::value, int> = 0>
    gdf_error operator()(gdf_column*  col_in, 
                    double       quant,
                    void*        t_erased_res,
                    gdf_context* ctxt,
                    hipStream_t stream = NULL)
    {
      return trampoline_exact<T, T>(col_in, cudf::interpolation::LOWER, quant, t_erased_res, ctxt, stream);
    }
  };

} // end of anonymous

namespace cudf {

gdf_error quantile_exact( gdf_column*         col_in,       // input column
                          interpolation       prec,         // interpolation method
                          double              q,            // requested quantile in [0,1]
                          gdf_scalar*         result,       // the result
                          gdf_context*        ctxt)         // context info
{
  GDF_REQUIRE(nullptr != col_in, GDF_DATASET_EMPTY);

  if (col_in->size == 0) {
     result->is_valid = false;
     return GDF_SUCCESS;
  }

  GDF_REQUIRE(nullptr != col_in->data, GDF_DATASET_EMPTY);
  GDF_REQUIRE(0 < col_in->size, GDF_DATASET_EMPTY);
  GDF_REQUIRE(nullptr == col_in->valid || 0 == col_in->null_count, GDF_VALIDITY_UNSUPPORTED);

  gdf_error ret = GDF_SUCCESS;
  result->dtype = GDF_FLOAT64;
  result->is_valid = false; // the scalar is not valid for error case

  ret = cudf::type_dispatcher(col_in->dtype,
                              trampoline_exact_functor{},
                              col_in, prec, q, &result->data, ctxt);

  if( ret == GDF_SUCCESS ) result->is_valid = true;
  return ret;
}

gdf_error quantile_approx(	gdf_column*  col_in,       // input column
                            double       q,            // requested quantile in [0,1]
                            gdf_scalar*  result,       // the result
                            gdf_context* ctxt)         // context info
{
  GDF_REQUIRE(nullptr != col_in, GDF_DATASET_EMPTY);
  
  if (col_in->size == 0) {
     result->is_valid = false;
     return GDF_SUCCESS;
  }

  GDF_REQUIRE(nullptr != col_in->data, GDF_DATASET_EMPTY);
  GDF_REQUIRE(0 < col_in->size, GDF_DATASET_EMPTY);
  GDF_REQUIRE(nullptr == col_in->valid || 0 == col_in->null_count, GDF_VALIDITY_UNSUPPORTED);

  gdf_error ret = GDF_SUCCESS;
  result->dtype = col_in->dtype;
  result->is_valid = false; // the scalar is not valid for error case

  ret = cudf::type_dispatcher(col_in->dtype,
                              trampoline_approx_functor{},
                              col_in, q, &result->data, ctxt);
  
  if( ret == GDF_SUCCESS ) result->is_valid = true;
  return ret;
}

} // namespace cudf
