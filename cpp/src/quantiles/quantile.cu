/*
 * Copyright (c) 2019-2020, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <memory>
#include <vector>

#include <cudf/copying.hpp>
#include <cudf/detail/gather.cuh>
#include <cudf/detail/nvtx/ranges.hpp>
#include <cudf/detail/sorting.hpp>
#include <cudf/table/table_view.hpp>
#include <cudf/types.hpp>
#include <cudf/utilities/error.hpp>
#include <quantiles/quantiles_util.hpp>

namespace cudf {
namespace detail {
template <bool exact, typename SortMapIterator>
struct quantile_functor {
  SortMapIterator ordered_indices;
  size_type size;
  std::vector<double> const& q;
  interpolation interp;
  bool retain_types;
  rmm::mr::device_memory_resource* mr;
  hipStream_t stream;

  template <typename T>
  std::enable_if_t<not std::is_arithmetic<T>::value, std::unique_ptr<column>> operator()(
    column_view const& input)
  {
    CUDF_FAIL("quantile does not support non-numeric types");
  }

  template <typename T>
  std::enable_if_t<std::is_arithmetic<T>::value, std::unique_ptr<column>> operator()(
    column_view const& input)
  {
    using Result = std::conditional_t<exact, double, T>;

    auto type   = data_type{type_to_id<Result>()};
    auto output = make_fixed_width_column(type, q.size(), mask_state::UNALLOCATED, stream, mr);

    if (output->size() == 0) { return output; }

    if (input.is_empty()) {
      auto mask = create_null_mask(output->size(), mask_state::ALL_NULL, stream, mr);
      output->set_null_mask(std::move(mask), output->size());
      return output;
    }

    auto d_input  = column_device_view::create(input);
    auto d_output = mutable_column_device_view::create(output->mutable_view());

    rmm::device_vector<double> q_device{q};

    auto sorted_data = thrust::make_permutation_iterator(input.data<T>(), ordered_indices);

    thrust::transform(q_device.begin(),
                      q_device.end(),
                      d_output->template begin<Result>(),
                      [sorted_data, interp = interp, size = size] __device__(double q) {
                        return select_quantile_data<Result>(sorted_data, size, q, interp);
                      });

    if (input.nullable()) {
      auto sorted_validity = thrust::make_transform_iterator(
        ordered_indices,
        [input = *d_input] __device__(size_type idx) { return input.is_valid_nocheck(idx); });

      rmm::device_buffer mask;
      size_type null_count;

      std::tie(mask, null_count) = valid_if(
        q_device.begin(),
        q_device.end(),
        [sorted_validity, interp = interp, size = size] __device__(double q) {
          return select_quantile_validity(sorted_validity, size, q, interp);
        },
        stream,
        mr);

      output->set_null_mask(std::move(mask), null_count);
    }

    return output;
  }
};

template <bool exact, typename SortMapIterator>
std::unique_ptr<column> quantile(column_view const& input,
                                 SortMapIterator ordered_indices,
                                 size_type size,
                                 std::vector<double> const& q,
                                 interpolation interp,
                                 bool retain_types,
                                 rmm::mr::device_memory_resource* mr,
                                 hipStream_t stream)
{
  auto functor = quantile_functor<exact, SortMapIterator>{
    ordered_indices, size, q, interp, retain_types, mr, stream};

  return type_dispatcher(input.type(), functor, input);
}

}  // namespace detail

std::unique_ptr<column> quantile(column_view const& input,
                                 std::vector<double> const& q,
                                 interpolation interp,
                                 column_view const& ordered_indices,
                                 bool exact,
                                 rmm::mr::device_memory_resource* mr)
{
  CUDF_FUNC_RANGE();

  if (ordered_indices.is_empty()) {
    if (exact) {
      return detail::quantile<true>(
        input, thrust::make_counting_iterator<size_type>(0), input.size(), q, interp, exact, mr, 0);
    } else {
      return detail::quantile<false>(
        input, thrust::make_counting_iterator<size_type>(0), input.size(), q, interp, exact, mr, 0);
    }

  } else {
    CUDF_EXPECTS(ordered_indices.type() == data_type{type_to_id<size_type>()},
                 "`ordered_indicies` type must be `INT32`.");

    if (exact) {
      return detail::quantile<true>(
        input, ordered_indices.data<size_type>(), ordered_indices.size(), q, interp, exact, mr, 0);
    } else {
      return detail::quantile<false>(
        input, ordered_indices.data<size_type>(), ordered_indices.size(), q, interp, exact, mr, 0);
    }
  }
}

}  // namespace cudf
