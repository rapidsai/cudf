#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2021-2022, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <quantiles/tdigest/tdigest_util.cuh>

#include <cudf/column/column_factories.hpp>
#include <cudf/column/column_view.hpp>
#include <cudf/copying.hpp>
#include <cudf/detail/concatenate.hpp>
#include <cudf/detail/copy.hpp>
#include <cudf/detail/get_value.cuh>
#include <cudf/detail/iterator.cuh>
#include <cudf/detail/merge.cuh>
#include <cudf/detail/sorting.hpp>
#include <cudf/detail/tdigest/tdigest.hpp>
#include <cudf/detail/utilities/cuda.cuh>
#include <cudf/lists/lists_column_view.hpp>
#include <cudf/utilities/span.hpp>

#include <rmm/device_uvector.hpp>
#include <rmm/exec_policy.hpp>

#include <thrust/advance.h>
#include <thrust/binary_search.h>
#include <thrust/distance.h>
#include <thrust/execution_policy.h>
#include <thrust/functional.h>
#include <thrust/iterator/constant_iterator.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/iterator/discard_iterator.h>
#include <thrust/iterator/transform_iterator.h>
#include <thrust/iterator/zip_iterator.h>
#include <thrust/merge.h>
#include <thrust/pair.h>
#include <thrust/reduce.h>
#include <thrust/remove.h>
#include <thrust/replace.h>
#include <thrust/scan.h>
#include <thrust/transform.h>
#include <thrust/tuple.h>

namespace cudf {
namespace tdigest {
namespace detail {

namespace {

// the most representative point within a cluster of similar
// values. {mean, weight}
// NOTE: Using a tuple here instead of a struct to take advantage of
// thrust zip iterators for output.
using centroid = thrust::tuple<double, double, bool>;

// make a centroid from a scalar with a weight of 1.
template <typename T>
struct make_centroid {
  column_device_view const col;

  centroid operator() __device__(size_type index) const
  {
    auto const is_valid = col.is_valid(index);
    auto const mean     = is_valid ? static_cast<double>(col.element<T>(index)) : 0.0;
    auto const weight   = is_valid ? 1.0 : 0.0;
    return {mean, weight, is_valid};
  }
};

// make a centroid from a scalar with a weight of 1. this functor
// assumes any value index it is passed is not null
template <typename T>
struct make_centroid_no_nulls {
  column_device_view const col;

  centroid operator() __device__(size_type index) const
  {
    return {static_cast<double>(col.element<T>(index)), 1.0, true};
  }
};

// make a centroid from an input stream of mean/weight values.
struct make_weighted_centroid {
  double const* mean;
  double const* weight;

  centroid operator() __device__(size_type index) { return {mean[index], weight[index], true}; }
};

// merge two centroids
struct merge_centroids {
  centroid operator() __device__(centroid const& lhs, centroid const& rhs) const
  {
    bool const lhs_valid = thrust::get<2>(lhs);
    bool const rhs_valid = thrust::get<2>(rhs);
    if (!lhs_valid && !rhs_valid) { return {0, 0, false}; }
    if (!lhs_valid) { return rhs; }
    if (!rhs_valid) { return lhs; }

    double const lhs_mean   = thrust::get<0>(lhs);
    double const rhs_mean   = thrust::get<0>(rhs);
    double const lhs_weight = thrust::get<1>(lhs);
    double const rhs_weight = thrust::get<1>(rhs);
    double const new_weight = lhs_weight + rhs_weight;
    return {(lhs_mean * lhs_weight + rhs_mean * rhs_weight) / new_weight, new_weight, true};
  }
};

/**
 * @brief A functor which returns the nearest cumulative weight in the grouped input stream prior to
 * the specified next weight limit.
 *
 * This functor assumes the weight for all scalars is simply 1. Under this assumption,
 * the nearest weight that will be <= the next limit is simply the nearest integer < the limit,
 * which we can get by just taking floor(next_limit).  For example if our next limit is 3.56, the
 * nearest whole number <= it is floor(3.56) == 3.
 */
struct nearest_value_scalar_weights_grouped {
  offset_type const* group_offsets;

  thrust::pair<double, int> operator() __device__(double next_limit, size_type group_index) const
  {
    double const f                   = floor(next_limit);
    auto const relative_weight_index = max(0, static_cast<int>(next_limit) - 1);
    auto const group_size            = group_offsets[group_index + 1] - group_offsets[group_index];
    return {f, relative_weight_index < group_size ? relative_weight_index : group_size - 1};
  }
};

/**
 * @brief A functor which returns the nearest cumulative weight in the input stream prior to the
 * specified next weight limit.
 *
 * This functor assumes the weight for all scalars is simply 1. Under this assumption,
 * the nearest weight that will be <= the next limit is simply the nearest integer < the limit,
 * which we can get by just taking floor(next_limit).  For example if our next limit is 3.56, the
 * nearest whole number <= it is floor(3.56) == 3.
 */
struct nearest_value_scalar_weights {
  size_type const input_size;

  thrust::pair<double, int> operator() __device__(double next_limit, size_type) const
  {
    double const f                   = floor(next_limit);
    auto const relative_weight_index = max(0, static_cast<int>(next_limit) - 1);
    return {f, relative_weight_index < input_size ? relative_weight_index : input_size - 1};
  }
};

/**
 * @brief A functor which returns the nearest cumulative weight in the input stream prior to the
 * specified next weight limit.
 *
 * This functor assumes we are dealing with grouped, sorted, weighted centroids.
 */
template <typename GroupOffsetsIter>
struct nearest_value_centroid_weights {
  double const* cumulative_weights;
  GroupOffsetsIter outer_offsets;    // groups
  offset_type const* inner_offsets;  // tdigests within a group

  thrust::pair<double, int> operator() __device__(double next_limit, size_type group_index) const
  {
    auto const tdigest_begin = outer_offsets[group_index];
    auto const tdigest_end   = outer_offsets[group_index + 1];
    auto const num_weights   = inner_offsets[tdigest_end] - inner_offsets[tdigest_begin];
    // NOTE: as it is today, this functor will never be called for any digests that are empty, but
    // I'll leave this check here for safety.
    if (num_weights == 0) { return thrust::pair<double, int>{0, 0}; }
    double const* group_cumulative_weights = cumulative_weights + inner_offsets[tdigest_begin];

    auto const index = ((thrust::lower_bound(thrust::seq,
                                             group_cumulative_weights,
                                             group_cumulative_weights + num_weights,
                                             next_limit)) -
                        group_cumulative_weights);

    return index == 0 ? thrust::pair<double, int>{0, 0}
                      : thrust::pair<double, int>{group_cumulative_weights[index - 1],
                                                  static_cast<int>(index) - 1};
  }
};

/**
 * @brief A functor which returns the cumulative input weight for a given index in a
 * set of grouped input values.
 *
 * This functor assumes the weight for all scalars is simply 1. Under this assumption,
 * the cumulative weight for a given value index I is simply I+1.
 */
struct cumulative_scalar_weight_grouped {
  cudf::device_span<size_type const> group_offsets;
  cudf::device_span<size_type const> group_labels;
  std::tuple<size_type, size_type, double> operator() __device__(size_type value_index) const
  {
    auto const group_index          = group_labels[value_index];
    auto const relative_value_index = value_index - group_offsets[group_index];
    return {group_index, relative_value_index, relative_value_index + 1};
  }
};

/**
 * @brief A functor which returns the cumulative input weight for a given index in a
 * set of input values.
 *
 * This functor assumes the weight for all scalars is simply 1. Under this assumption,
 * the cumulative weight for a given value index I is simply I+1.
 */
struct cumulative_scalar_weight {
  std::tuple<size_type, size_type, double> operator() __device__(size_type value_index) const
  {
    return {0, value_index, value_index + 1};
  }
};

/**
 * @brief A functor which returns the cumulative input weight for a given index in a
 * set of grouped input centroids.
 *
 * This functor assumes we are dealing with grouped, weighted centroids.
 */
template <typename GroupLabelsIter, typename GroupOffsetsIter>
struct cumulative_centroid_weight {
  double const* cumulative_weights;
  GroupLabelsIter group_labels;
  GroupOffsetsIter outer_offsets;                      // groups
  cudf::device_span<offset_type const> inner_offsets;  // tdigests with a group

  std::tuple<size_type, size_type, double> operator() __device__(size_type value_index) const
  {
    auto const tdigest_index =
      static_cast<size_type>(
        thrust::upper_bound(thrust::seq, inner_offsets.begin(), inner_offsets.end(), value_index) -
        inner_offsets.begin()) -
      1;
    auto const group_index                 = group_labels[tdigest_index];
    auto const first_tdigest_index         = outer_offsets[group_index];
    auto const first_weight_index          = inner_offsets[first_tdigest_index];
    auto const relative_value_index        = value_index - first_weight_index;
    double const* group_cumulative_weights = cumulative_weights + first_weight_index;

    return {group_index, relative_value_index, group_cumulative_weights[relative_value_index]};
  }
};

// retrieve group info (total weight, size, start offset) of scalar inputs by group index.
struct scalar_group_info_grouped {
  size_type const* group_valid_counts;
  offset_type const* group_offsets;

  __device__ thrust::tuple<double, size_type, size_type> operator()(size_type group_index) const
  {
    return {static_cast<double>(group_valid_counts[group_index]),
            group_offsets[group_index + 1] - group_offsets[group_index],
            group_offsets[group_index]};
  }
};

// retrieve group info (total weight, size, start offset) of scalar inputs
struct scalar_group_info {
  double const total_weight;
  size_type const size;

  __device__ thrust::tuple<double, size_type, size_type> operator()(size_type) const
  {
    return {total_weight, size, 0};
  }
};

// retrieve group info of centroid inputs by group index
template <typename GroupOffsetsIter>
struct centroid_group_info {
  double const* cumulative_weights;
  GroupOffsetsIter outer_offsets;
  offset_type const* inner_offsets;

  __device__ thrust::tuple<double, size_type, size_type> operator()(size_type group_index) const
  {
    // if there's no weights in this group of digests at all, return 0.
    auto const group_start       = inner_offsets[outer_offsets[group_index]];
    auto const group_end         = inner_offsets[outer_offsets[group_index + 1]];
    auto const num_weights       = group_end - group_start;
    auto const last_weight_index = group_end - 1;
    return num_weights == 0
             ? thrust::tuple<double, size_type, size_type>{0, num_weights, group_start}
             : thrust::tuple<double, size_type, size_type>{
                 cumulative_weights[last_weight_index], num_weights, group_start};
  }
};

struct tdigest_min {
  __device__ double operator()(thrust::tuple<double, size_type> const& t) const
  {
    auto const min  = thrust::get<0>(t);
    auto const size = thrust::get<1>(t);
    return size > 0 ? min : std::numeric_limits<double>::max();
  }
};

struct tdigest_max {
  __device__ double operator()(thrust::tuple<double, size_type> const& t) const
  {
    auto const max  = thrust::get<0>(t);
    auto const size = thrust::get<1>(t);
    return size > 0 ? max : std::numeric_limits<double>::lowest();
  }
};

// a monotonically increasing scale function which produces a distribution
// of centroids that is more densely packed in the middle of the input
// than at the ends.
__device__ double scale_func_k1(double quantile, double delta_norm)
{
  double k = delta_norm * asin(2.0 * quantile - 1.0);
  k += 1.0;
  double const q = (sin(k / delta_norm) + 1.0) / 2.0;
  return q;
}

// convert a single-row tdigest column to a scalar.
std::unique_ptr<scalar> to_tdigest_scalar(std::unique_ptr<column>&& tdigest,
                                          rmm::cuda_stream_view stream,
                                          rmm::mr::device_memory_resource* mr)
{
  CUDF_EXPECTS(tdigest->size() == 1,
               "Encountered invalid tdigest column when converting to scalar");
  auto contents = tdigest->release();
  return std::make_unique<struct_scalar>(table(std::move(contents.children)), true, stream, mr);
}

/**
 * @brief Compute a set of cluster limits (brackets, essentially) for a
 * given tdigest based on the specified delta and the total weight of values
 * to be added.
 *
 * The number of clusters generated will always be <= delta_, where delta_ is
 * a reasonably small number likely << 10000.
 *
 * Each input group gets an independent set of clusters generated. 1 thread
 * per group.
 *
 * This kernel is called in a two-pass style.  Once to compute the per-group
 * cluster sizes and total # of clusters, and once to compute the actual
 * weight limits per cluster.
 *
 * @param delta               tdigest compression level
 * @param num_groups          The number of input groups
 * @param nearest_weight      A functor which returns the nearest weight in the input
 * stream that falls before our current cluster limit
 * @param group_info          A functor which returns the info for the specified group (total
 * weight, size and start offset)
 * @param group_cluster_wl    Output.  The set of cluster weight limits for each group.
 * @param group_num_clusters  Output.  The number of output clusters for each input group.
 * @param group_cluster_offsets  Offsets per-group to the start of it's clusters
 * @param has_nulls Whether or not the input contains nulls
 *
 */

template <typename GroupInfo, typename NearestWeightFunc, typename CumulativeWeight>
__global__ void generate_cluster_limits_kernel(int delta,
                                               size_type num_groups,
                                               NearestWeightFunc nearest_weight,
                                               GroupInfo group_info,
                                               CumulativeWeight cumulative_weight,
                                               double* group_cluster_wl,
                                               size_type* group_num_clusters,
                                               offset_type const* group_cluster_offsets,
                                               bool has_nulls)
{
  int const tid = threadIdx.x + blockIdx.x * blockDim.x;

  auto const group_index = tid;
  if (group_index >= num_groups) { return; }

  // we will generate at most delta clusters.
  double const delta_norm = static_cast<double>(delta) / (2.0 * M_PI);
  double total_weight;
  size_type group_size, group_start;
  thrust::tie(total_weight, group_size, group_start) = group_info(group_index);

  // start at the correct place based on our cluster offset.
  double* cluster_wl =
    group_cluster_wl ? group_cluster_wl + group_cluster_offsets[group_index] : nullptr;

  // a group with nothing in it.
  group_num_clusters[group_index] = 0;
  if (total_weight <= 0) {
    // if the input contains nulls we can potentially have a group that generates no
    // clusters because -all- of the input values are null.  in that case, the reduce_by_key call
    // in the tdigest generation step will need a location to store the unused reduction value for
    // that group of nulls. these "stubs" will be postprocessed out afterwards.
    if (has_nulls) { group_num_clusters[group_index] = 1; }
    return;
  }

  double cur_limit        = 0.0;
  double cur_weight       = 0.0;
  double next_limit       = -1.0;
  int last_inserted_index = -1;  // group-relative index into the input stream

  // compute the first cluster limit
  double nearest_w;
  int nearest_w_index;  // group-relative index into the input stream
  while (true) {
    cur_weight = next_limit < 0 ? 0 : max(cur_weight + 1, nearest_w);
    if (cur_weight >= total_weight) { break; }

    // based on where we are closing the cluster off (not including the incoming weight),
    // compute the next cluster limit
    double const quantile = cur_weight / total_weight;
    next_limit            = total_weight * scale_func_k1(quantile, delta_norm);

    // if the next limit is < the cur limit, we're past the end of the distribution, so we're done.
    if (next_limit <= cur_limit) {
      if (cluster_wl) { cluster_wl[group_num_clusters[group_index]] = total_weight; }
      group_num_clusters[group_index]++;
      break;
    }

    // compute the weight we will be at in the input values just before closing off the current
    // cluster (because adding the next value will cross the current limit).
    // NOTE: can't use structured bindings here.
    thrust::tie(nearest_w, nearest_w_index) = nearest_weight(next_limit, group_index);

    // because of the way the scale functions work, it is possible to generate clusters
    // in such a way that we end up with "gaps" where there are no input values that
    // fall into a given cluster.  An example would be this:
    //
    // cluster weight limits = 0.00003, 1.008, 3.008
    //
    // input values(weight) = A(1), B(2), C(3)
    //
    // naively inserting these values into the clusters simply by taking a lower_bound,
    // we would get the following distribution of input values into those 3 clusters.
    //  (), (A), (B,C)
    //
    // whereas what we really want is:
    //
    //  (A), (B), (C)
    //
    // to fix this, we will artificially adjust the output cluster limits to guarantee
    // at least 1 input value will be put in each cluster during the reduction step.
    // this does not affect final centroid results as we still use the "real" weight limits
    // to compute subsequent clusters - the purpose is only to allow cluster selection
    // during the reduction step to be trivial.
    //
    double adjusted_next_limit = next_limit;
    int adjusted_w_index       = nearest_w_index;
    if ((last_inserted_index < 0) ||  // if we haven't inserted anything yet
        (nearest_w_index ==
         last_inserted_index)) {  // if we land in the same bucket as the previous cap

      // force the value into this bucket
      adjusted_w_index = (last_inserted_index == group_size - 1)
                           ? last_inserted_index
                           : max(adjusted_w_index, last_inserted_index + 1);

      // the "adjusted" cluster limit must be high enough so that this value will fall in the
      // bucket. NOTE: cumulative_weight expects an absolute index into the input value stream, not
      // a group-relative index
      [[maybe_unused]] auto [r, i, adjusted_w] = cumulative_weight(adjusted_w_index + group_start);
      adjusted_next_limit                      = max(next_limit, adjusted_w);

      // update the weight with our adjusted value.
      nearest_w = adjusted_w;
    }
    if (cluster_wl) { cluster_wl[group_num_clusters[group_index]] = adjusted_next_limit; }
    last_inserted_index = adjusted_w_index;

    group_num_clusters[group_index]++;
    cur_limit = next_limit;
  }
}

/**
 * @brief Compute a set of cluster limits (brackets, essentially) for a
 * given tdigest based on the specified delta and the total weight of values
 * to be added.
 *
 * The number of clusters generated will always be <= delta_, where delta_ is
 * a reasonably small number likely << 10000.
 *
 * Each input group gets an independent set of clusters generated.
 *
 * @param delta_             tdigest compression level
 * @param num_groups         The number of input groups
 * @param nearest_weight     A functor which returns the nearest weight in the input
 * stream that falls before our current cluster limit
 * @param group_info         A functor which returns the info for the specified group (total weight,
 * size and start offset)
 * @param has_nulls          Whether or not the input data contains nulls
 * @param stream CUDA stream used for device memory operations and kernel launches.
 * @param mr Device memory resource used to allocate the returned column's device memory
 *
 * @returns A tuple containing the set of cluster weight limits for each group, a set of
 * list-style offsets indicating group sizes, and the total number of clusters
 */
template <typename GroupInfo, typename NearestWeight, typename CumulativeWeight>
std::tuple<rmm::device_uvector<double>, std::unique_ptr<column>, size_type>
generate_group_cluster_info(int delta,
                            size_type num_groups,
                            NearestWeight nearest_weight,
                            GroupInfo group_info,
                            CumulativeWeight cumulative_weight,
                            bool has_nulls,
                            rmm::cuda_stream_view stream,
                            rmm::mr::device_memory_resource* mr)
{
  constexpr size_type block_size = 256;
  cudf::detail::grid_1d const grid(num_groups, block_size);

  // compute number of clusters per group
  // each thread computes 1 set of clusters (# of cluster sets == # of groups)
  rmm::device_uvector<size_type> group_num_clusters(num_groups, stream);
  generate_cluster_limits_kernel<<<grid.num_blocks, block_size, 0, stream.value()>>>(
    delta,
    num_groups,
    nearest_weight,
    group_info,
    cumulative_weight,
    nullptr,
    group_num_clusters.begin(),
    nullptr,
    has_nulls);

  // generate group cluster offsets (where the clusters for a given group start and end)
  auto group_cluster_offsets = cudf::make_numeric_column(
    data_type{type_id::INT32}, num_groups + 1, mask_state::UNALLOCATED, stream, mr);
  auto cluster_size = cudf::detail::make_counting_transform_iterator(
    0, [group_num_clusters = group_num_clusters.begin(), num_groups] __device__(size_type index) {
      return index == num_groups ? 0 : group_num_clusters[index];
    });
  thrust::exclusive_scan(rmm::exec_policy(stream),
                         cluster_size,
                         cluster_size + num_groups + 1,
                         group_cluster_offsets->mutable_view().begin<offset_type>(),
                         0);

  // total # of clusters
  offset_type total_clusters =
    cudf::detail::get_value<offset_type>(group_cluster_offsets->view(), num_groups, stream);

  // fill in the actual cluster weight limits
  rmm::device_uvector<double> group_cluster_wl(total_clusters, stream);
  generate_cluster_limits_kernel<<<grid.num_blocks, block_size, 0, stream.value()>>>(
    delta,
    num_groups,
    nearest_weight,
    group_info,
    cumulative_weight,
    group_cluster_wl.begin(),
    group_num_clusters.begin(),
    group_cluster_offsets->view().begin<offset_type>(),
    has_nulls);

  return {std::move(group_cluster_wl),
          std::move(group_cluster_offsets),
          static_cast<size_type>(total_clusters)};
}

std::unique_ptr<column> build_output_column(size_type num_rows,
                                            std::unique_ptr<column>&& means,
                                            std::unique_ptr<column>&& weights,
                                            std::unique_ptr<column>&& offsets,
                                            std::unique_ptr<column>&& min_col,
                                            std::unique_ptr<column>&& max_col,
                                            bool has_nulls,
                                            rmm::cuda_stream_view stream,
                                            rmm::mr::device_memory_resource* mr)
{
  // whether or not this weight is a stub
  auto is_stub_weight = [weights = weights->view().begin<double>()] __device__(size_type i) {
    return weights[i] == 0;
  };
  // whether or not this particular tdigest is a stub
  auto is_stub_digest = [offsets = offsets->view().begin<offset_type>(), is_stub_weight] __device__(
                          size_type i) { return is_stub_weight(offsets[i]) ? 1 : 0; };

  size_type const num_stubs = [&]() {
    if (!has_nulls) { return 0; }
    auto iter = cudf::detail::make_counting_transform_iterator(0, is_stub_digest);
    return thrust::reduce(rmm::exec_policy(stream), iter, iter + num_rows);
  }();

  // if there are no stub tdigests, we can return immediately.
  if (num_stubs == 0) {
    return cudf::tdigest::detail::make_tdigest_column(num_rows,
                                                      std::move(means),
                                                      std::move(weights),
                                                      std::move(offsets),
                                                      std::move(min_col),
                                                      std::move(max_col),
                                                      stream,
                                                      mr);
  }

  // otherwise we need to strip out the stubs.
  auto remove_stubs = [&](column_view const& col, size_type num_stubs) {
    auto result = cudf::make_numeric_column(
      data_type{type_id::FLOAT64}, col.size() - num_stubs, mask_state::UNALLOCATED, stream, mr);
    thrust::remove_copy_if(rmm::exec_policy(stream),
                           col.begin<double>(),
                           col.end<double>(),
                           thrust::make_counting_iterator(0),
                           result->mutable_view().begin<double>(),
                           is_stub_weight);
    return result;
  };
  // remove from the means and weights column
  auto _means   = remove_stubs(*means, num_stubs);
  auto _weights = remove_stubs(*weights, num_stubs);

  // adjust offsets.
  rmm::device_uvector<offset_type> sizes(num_rows, stream);
  thrust::transform(rmm::exec_policy(stream),
                    thrust::make_counting_iterator(0),
                    thrust::make_counting_iterator(0) + num_rows,
                    sizes.begin(),
                    [offsets = offsets->view().begin<offset_type>()] __device__(size_type i) {
                      return offsets[i + 1] - offsets[i];
                    });
  auto iter = cudf::detail::make_counting_transform_iterator(
    0, [sizes = sizes.begin(), is_stub_digest, num_rows] __device__(size_type i) {
      return i == num_rows || is_stub_digest(i) ? 0 : sizes[i];
    });
  thrust::exclusive_scan(rmm::exec_policy(stream),
                         iter,
                         iter + num_rows + 1,
                         offsets->mutable_view().begin<offset_type>(),
                         0);

  // assemble final column
  return cudf::tdigest::detail::make_tdigest_column(num_rows,
                                                    std::move(_means),
                                                    std::move(_weights),
                                                    std::move(offsets),
                                                    std::move(min_col),
                                                    std::move(max_col),
                                                    stream,
                                                    mr);
}

/**
 * @brief Compute a column of tdigests.
 *
 * Assembles the output tdigest column based on the specified delta, a stream of
 * input values (either scalar or centroids), and an assortment of per-group
 * clustering information.
 *
 * This function is effectively just a reduce_by_key that performs a reduction
 * from input values -> centroid clusters as defined by the the cluster weight
 * boundaries.
 *
 * @param delta              tdigest compression level
 * @param values_begin       Beginning of the range of input values.
 * @param values_end         End of the range of input values.
 * @param cumulative_weight  Functor which returns cumulative weight and group information for
 * an absolute input value index.
 * @param min_col            Column containing the minimum value per group.
 * @param max_col            Column containing the maximum value per group.
 * @param group_cluster_wl   Cluster weight limits for each group.
 * @param group_cluster_offsets R-value reference of offsets into the cluster weight limits.
 * @param total_clusters     Total number of clusters in all groups.
 * @param has_nulls          Whether or not the input contains nulls
 * @param stream CUDA stream used for device memory operations and kernel launches.
 * @param mr Device memory resource used to allocate the returned column's device memory
 *
 * @returns A tdigest column with 1 row per output tdigest.
 */
template <typename CentroidIter, typename CumulativeWeight>
std::unique_ptr<column> compute_tdigests(int delta,
                                         CentroidIter centroids_begin,
                                         CentroidIter centroids_end,
                                         CumulativeWeight group_cumulative_weight,
                                         std::unique_ptr<column>&& min_col,
                                         std::unique_ptr<column>&& max_col,
                                         rmm::device_uvector<double> const& group_cluster_wl,
                                         std::unique_ptr<column>&& group_cluster_offsets,
                                         size_type total_clusters,
                                         bool has_nulls,
                                         rmm::cuda_stream_view stream,
                                         rmm::mr::device_memory_resource* mr)
{
  // the output for each group is a column of data that represents the tdigest. since we want 1 row
  // per group, each row will be a list the length of the tdigest for that group. so our output
  // column is of the form:
  // struct {
  //   centroids for the digest
  //   list {
  //     struct {
  //       double    // mean
  //       double    // weight
  //     }
  //   }
  //   double       // min
  //   double       // max
  // }
  //
  if (total_clusters == 0) { return cudf::tdigest::detail::make_empty_tdigest_column(stream, mr); }

  // each input group represents an individual tdigest.  within each tdigest, we want the keys
  // to represent cluster indices (for example, if a tdigest had 100 clusters, the keys should fall
  // into the range 0-99).  But since we have multiple tdigests, we need to keep the keys unique
  // between the groups, so we add our group start offset.
  auto keys = thrust::make_transform_iterator(
    thrust::make_counting_iterator(0),
    [delta,
     group_cluster_wl      = group_cluster_wl.data(),
     group_cluster_offsets = group_cluster_offsets->view().begin<offset_type>(),
     group_cumulative_weight] __device__(size_type value_index) -> size_type {
      // get group index, relative value index within the group and cumulative weight.
      [[maybe_unused]] auto [group_index, relative_value_index, cumulative_weight] =
        group_cumulative_weight(value_index);

      auto const num_clusters =
        group_cluster_offsets[group_index + 1] - group_cluster_offsets[group_index];
      if (num_clusters == 0) { return group_cluster_offsets[group_index]; }

      // compute start of cluster weight limits for this group
      double const* weight_limits = group_cluster_wl + group_cluster_offsets[group_index];

      // local cluster index
      size_type const group_cluster_index =
        min(num_clusters - 1,
            static_cast<size_type>(
              thrust::lower_bound(
                thrust::seq, weight_limits, weight_limits + num_clusters, cumulative_weight) -
              weight_limits));

      // add the cluster offset to generate a globally unique key
      return group_cluster_index + group_cluster_offsets[group_index];
    });

  // mean and weight data
  auto centroid_means = cudf::make_numeric_column(
    data_type{type_id::FLOAT64}, total_clusters, mask_state::UNALLOCATED, stream, mr);
  auto centroid_weights = cudf::make_numeric_column(
    data_type{type_id::FLOAT64}, total_clusters, mask_state::UNALLOCATED, stream, mr);
  // reduce the centroids down by key.
  cudf::mutable_column_view mean_col(*centroid_means);
  cudf::mutable_column_view weight_col(*centroid_weights);

  // reduce the centroids into the clusters
  auto output = thrust::make_zip_iterator(thrust::make_tuple(
    mean_col.begin<double>(), weight_col.begin<double>(), thrust::make_discard_iterator()));

  auto const num_values = std::distance(centroids_begin, centroids_end);
  thrust::reduce_by_key(rmm::exec_policy(stream),
                        keys,
                        keys + num_values,                // keys
                        centroids_begin,                  // values
                        thrust::make_discard_iterator(),  // key output
                        output,                           // output
                        thrust::equal_to{},               // key equality check
                        merge_centroids{});

  // create final tdigest column
  return build_output_column(group_cluster_offsets->size() - 1,
                             std::move(centroid_means),
                             std::move(centroid_weights),
                             std::move(group_cluster_offsets),
                             std::move(min_col),
                             std::move(max_col),
                             has_nulls,
                             stream,
                             mr);
}

// return the min/max value of scalar inputs by group index
template <typename T>
struct get_scalar_minmax_grouped {
  column_device_view const col;
  device_span<size_type const> group_offsets;
  size_type const* group_valid_counts;

  __device__ thrust::tuple<double, double> operator()(size_type group_index)
  {
    auto const valid_count = group_valid_counts[group_index];
    return valid_count > 0
             ? thrust::make_tuple(
                 static_cast<double>(col.element<T>(group_offsets[group_index])),
                 static_cast<double>(col.element<T>(group_offsets[group_index] + valid_count - 1)))
             : thrust::make_tuple(0.0, 0.0);
  }
};

// return the min/max value of scalar inputs
template <typename T>
struct get_scalar_minmax {
  column_device_view const col;
  size_type const valid_count;

  __device__ thrust::tuple<double, double> operator()(size_type)
  {
    return valid_count > 0
             ? thrust::make_tuple(static_cast<double>(col.element<T>(0)),
                                  static_cast<double>(col.element<T>(valid_count - 1)))
             : thrust::make_tuple(0.0, 0.0);
  }
};

struct typed_group_tdigest {
  template <typename T,
            std::enable_if_t<cudf::is_numeric<T>() || cudf::is_fixed_point<T>()>* = nullptr>
  std::unique_ptr<column> operator()(column_view const& col,
                                     cudf::device_span<size_type const> group_offsets,
                                     cudf::device_span<size_type const> group_labels,
                                     cudf::device_span<size_type const> group_valid_counts,
                                     size_type num_groups,
                                     int delta,
                                     rmm::cuda_stream_view stream,
                                     rmm::mr::device_memory_resource* mr)
  {
    // first, generate cluster weight information for each input group
    auto [group_cluster_wl, group_cluster_offsets, total_clusters] = generate_group_cluster_info(
      delta,
      num_groups,
      nearest_value_scalar_weights_grouped{group_offsets.begin()},
      scalar_group_info_grouped{group_valid_counts.begin(), group_offsets.begin()},
      cumulative_scalar_weight_grouped{group_offsets, group_labels},
      col.null_count() > 0,
      stream,
      mr);

    // device column view. handy because the .element() function
    // automatically handles fixed-point conversions for us
    auto d_col = cudf::column_device_view::create(col, stream);

    // compute min and max columns
    auto min_col = cudf::make_numeric_column(
      data_type{type_id::FLOAT64}, num_groups, mask_state::UNALLOCATED, stream, mr);
    auto max_col = cudf::make_numeric_column(
      data_type{type_id::FLOAT64}, num_groups, mask_state::UNALLOCATED, stream, mr);
    thrust::transform(
      rmm::exec_policy(stream),
      thrust::make_counting_iterator(0),
      thrust::make_counting_iterator(0) + num_groups,
      thrust::make_zip_iterator(thrust::make_tuple(min_col->mutable_view().begin<double>(),
                                                   max_col->mutable_view().begin<double>())),
      get_scalar_minmax_grouped<T>{*d_col, group_offsets, group_valid_counts.begin()});

    // for simple input values, the "centroids" all have a weight of 1.
    auto scalar_to_centroid =
      cudf::detail::make_counting_transform_iterator(0, make_centroid<T>{*d_col});

    // generate the final tdigest
    return compute_tdigests(delta,
                            scalar_to_centroid,
                            scalar_to_centroid + col.size(),
                            cumulative_scalar_weight_grouped{group_offsets, group_labels},
                            std::move(min_col),
                            std::move(max_col),
                            group_cluster_wl,
                            std::move(group_cluster_offsets),
                            total_clusters,
                            col.null_count() > 0,
                            stream,
                            mr);
  }

  template <typename T,
            typename... Args,
            std::enable_if_t<!cudf::is_numeric<T>() && !cudf::is_fixed_point<T>()>* = nullptr>
  std::unique_ptr<column> operator()(Args&&...)
  {
    CUDF_FAIL("Non-numeric type in group_tdigest");
  }
};

struct typed_reduce_tdigest {
  // this function assumes col is sorted in ascending order with nulls at the end
  template <
    typename T,
    typename std::enable_if_t<cudf::is_numeric<T>() || cudf::is_fixed_point<T>()>* = nullptr>
  std::unique_ptr<scalar> operator()(column_view const& col,
                                     int delta,
                                     rmm::cuda_stream_view stream,
                                     rmm::mr::device_memory_resource* mr)
  {
    // treat this the same as the groupby path with a single group.  Note:  even though
    // there is only 1 group there are still multiple keys within the group that represent
    // the clustering of (N input values) -> (1 output centroid), so the final computation
    // remains a reduce_by_key() and not a reduce().
    //
    // additionally we get a few optimizations.
    // - since we only ever have 1 "group" that is sorted with nulls at the end,
    //   we can simply process just the non-null values and act as if the column
    //   is non-nullable, allowing us to process fewer values than if we were doing a groupby.
    //
    // - several of the functors used during the reduction are cheaper than during a groupby.

    auto const valid_count = col.size() - col.null_count();

    // first, generate cluster weight information for each input group
    auto [cluster_wl, cluster_offsets, total_clusters] =
      generate_group_cluster_info(delta,
                                  1,
                                  nearest_value_scalar_weights{valid_count},
                                  scalar_group_info{static_cast<double>(valid_count), valid_count},
                                  cumulative_scalar_weight{},
                                  false,
                                  stream,
                                  mr);

    // device column view. handy because the .element() function
    // automatically handles fixed-point conversions for us
    auto d_col = cudf::column_device_view::create(col, stream);

    // compute min and max columns
    auto min_col = cudf::make_numeric_column(
      data_type{type_id::FLOAT64}, 1, mask_state::UNALLOCATED, stream, mr);
    auto max_col = cudf::make_numeric_column(
      data_type{type_id::FLOAT64}, 1, mask_state::UNALLOCATED, stream, mr);
    thrust::transform(
      rmm::exec_policy(stream),
      thrust::make_counting_iterator(0),
      thrust::make_counting_iterator(0) + 1,
      thrust::make_zip_iterator(thrust::make_tuple(min_col->mutable_view().begin<double>(),
                                                   max_col->mutable_view().begin<double>())),
      get_scalar_minmax<T>{*d_col, valid_count});

    // for simple input values, the "centroids" all have a weight of 1.
    auto scalar_to_centroid =
      cudf::detail::make_counting_transform_iterator(0, make_centroid_no_nulls<T>{*d_col});

    // generate the final tdigest and wrap it in a struct_scalar
    return to_tdigest_scalar(compute_tdigests(delta,
                                              scalar_to_centroid,
                                              scalar_to_centroid + valid_count,
                                              cumulative_scalar_weight{},
                                              std::move(min_col),
                                              std::move(max_col),
                                              cluster_wl,
                                              std::move(cluster_offsets),
                                              total_clusters,
                                              false,
                                              stream,
                                              mr),
                             stream,
                             mr);
  }

  template <
    typename T,
    typename... Args,
    typename std::enable_if_t<!cudf::is_numeric<T>() && !cudf::is_fixed_point<T>()>* = nullptr>
  std::unique_ptr<scalar> operator()(Args&&...)
  {
    CUDF_FAIL("Non-numeric type in group_tdigest");
  }
};

// utility for merge_tdigests.
template <typename GroupOffsetsIter>
struct group_num_weights_func {
  GroupOffsetsIter outer_offsets;
  size_type const* inner_offsets;

  __device__ size_type operator()(size_type group_index)
  {
    auto const tdigest_begin = outer_offsets[group_index];
    auto const tdigest_end   = outer_offsets[group_index + 1];
    return inner_offsets[tdigest_end] - inner_offsets[tdigest_begin];
  }
};

// utility for merge_tdigests.
struct group_is_empty {
  __device__ bool operator()(size_type group_size) { return group_size == 0; }
};

// utility for merge_tdigests.
template <typename GroupLabelsIter>
struct group_key_func {
  GroupLabelsIter group_labels;
  size_type const* inner_offsets;
  size_type num_inner_offsets;

  __device__ size_type operator()(size_type index)
  {
    // what -original- tdigest index this absolute index corresponds to
    auto const iter = thrust::prev(
      thrust::upper_bound(thrust::seq, inner_offsets, inner_offsets + num_inner_offsets, index));
    auto const tdigest_index = thrust::distance(inner_offsets, iter);

    // what group index the original tdigest belongs to
    return group_labels[tdigest_index];
  }
};

template <typename HGroupOffsetIter, typename GroupOffsetIter, typename GroupLabelIter>
std::unique_ptr<column> merge_tdigests(tdigest_column_view const& tdv,
                                       HGroupOffsetIter h_outer_offsets,
                                       GroupOffsetIter group_offsets,
                                       GroupLabelIter group_labels,
                                       size_t num_group_labels,
                                       size_type num_groups,
                                       int max_centroids,
                                       rmm::cuda_stream_view stream,
                                       rmm::mr::device_memory_resource* mr)
{
  // thrust::merge and thrust::merge_by_key don't provide what we need.  What we would need is an
  // algorithm like a super-merge that takes two layers of keys: one which identifies the outer
  // grouping of tdigests, and one which identifies the inner groupings of the tdigests within the
  // outer groups.
  // TODO: investigate replacing the iterative merge with a single stable_sort_by_key.

  // bring tdigest offsets back to the host
  auto tdigest_offsets = tdv.centroids().offsets();
  std::vector<offset_type> h_inner_offsets(tdigest_offsets.size());
  hipMemcpyAsync(h_inner_offsets.data(),
                  tdigest_offsets.begin<offset_type>(),
                  sizeof(offset_type) * tdigest_offsets.size(),
                  hipMemcpyDeviceToHost,
                  stream);

  stream.synchronize();

  // extract all means and weights into a table
  cudf::table_view tdigests_unsliced({tdv.means(), tdv.weights()});

  // generate the merged (but not yet compressed) tdigests for each group.
  std::vector<std::unique_ptr<table>> tdigests;
  tdigests.reserve(num_groups);
  std::transform(h_outer_offsets,
                 h_outer_offsets + num_groups,
                 std::next(h_outer_offsets),
                 std::back_inserter(tdigests),
                 [&](auto tdigest_start, auto tdigest_end) {
                   // the range of tdigests in this group
                   auto const num_tdigests = tdigest_end - tdigest_start;

                   // slice each tdigest from the input
                   std::vector<table_view> unmerged_tdigests;
                   unmerged_tdigests.reserve(num_tdigests);
                   auto offset_iter = std::next(h_inner_offsets.begin(), tdigest_start);
                   std::transform(
                     offset_iter,
                     offset_iter + num_tdigests,
                     std::next(offset_iter),
                     std::back_inserter(unmerged_tdigests),
                     [&](size_type start, size_type end) {
                       return cudf::detail::slice(tdigests_unsliced, {start, end}, stream);
                     });

                   // merge
                   return cudf::detail::merge(unmerged_tdigests,
                                              {0},
                                              {order::ASCENDING},
                                              {},
                                              stream,
                                              rmm::mr::get_current_device_resource());
                 });

  // generate min and max values
  auto merged_min_col = cudf::make_numeric_column(
    data_type{type_id::FLOAT64}, num_groups, mask_state::UNALLOCATED, stream, mr);
  auto min_iter =
    thrust::make_transform_iterator(thrust::make_zip_iterator(thrust::make_tuple(
                                      tdv.min_begin(), cudf::tdigest::detail::size_begin(tdv))),
                                    tdigest_min{});
  thrust::reduce_by_key(rmm::exec_policy(stream),
                        group_labels,
                        group_labels + num_group_labels,
                        min_iter,
                        thrust::make_discard_iterator(),
                        merged_min_col->mutable_view().begin<double>(),
                        thrust::equal_to{},  // key equality check
                        thrust::minimum{});

  auto merged_max_col = cudf::make_numeric_column(
    data_type{type_id::FLOAT64}, num_groups, mask_state::UNALLOCATED, stream, mr);
  auto max_iter =
    thrust::make_transform_iterator(thrust::make_zip_iterator(thrust::make_tuple(
                                      tdv.max_begin(), cudf::tdigest::detail::size_begin(tdv))),
                                    tdigest_max{});
  thrust::reduce_by_key(rmm::exec_policy(stream),
                        group_labels,
                        group_labels + num_group_labels,
                        max_iter,
                        thrust::make_discard_iterator(),
                        merged_max_col->mutable_view().begin<double>(),
                        thrust::equal_to{},  // key equality check
                        thrust::maximum{});

  // for any empty groups, set the min and max to be 0. not technically necessary but it makes
  // testing simpler.
  auto group_num_weights = cudf::detail::make_counting_transform_iterator(
    0,
    group_num_weights_func<decltype(group_offsets)>{group_offsets,
                                                    tdigest_offsets.begin<size_type>()});
  thrust::replace_if(rmm::exec_policy(stream),
                     merged_min_col->mutable_view().begin<double>(),
                     merged_min_col->mutable_view().end<double>(),
                     group_num_weights,
                     group_is_empty{},
                     0);
  thrust::replace_if(rmm::exec_policy(stream),
                     merged_max_col->mutable_view().begin<double>(),
                     merged_max_col->mutable_view().end<double>(),
                     group_num_weights,
                     group_is_empty{},
                     0);

  // concatenate all the merged tdigests back into one table.
  std::vector<table_view> tdigest_views;
  tdigest_views.reserve(num_groups);
  std::transform(tdigests.begin(),
                 tdigests.end(),
                 std::back_inserter(tdigest_views),
                 [](std::unique_ptr<table> const& t) { return t->view(); });
  auto merged = cudf::detail::concatenate(tdigest_views, stream);

  // generate cumulative weights
  auto merged_weights     = merged->get_column(1).view();
  auto cumulative_weights = cudf::make_numeric_column(
    data_type{type_id::FLOAT64}, merged_weights.size(), mask_state::UNALLOCATED);
  auto keys = cudf::detail::make_counting_transform_iterator(
    0,
    group_key_func<decltype(group_labels)>{
      group_labels, tdigest_offsets.begin<size_type>(), tdigest_offsets.size()});
  thrust::inclusive_scan_by_key(rmm::exec_policy(stream),
                                keys,
                                keys + cumulative_weights->size(),
                                merged_weights.begin<double>(),
                                cumulative_weights->mutable_view().begin<double>());

  auto const delta = max_centroids;

  // generate cluster info
  auto [group_cluster_wl, group_cluster_offsets, total_clusters] = generate_group_cluster_info(
    delta,
    num_groups,
    nearest_value_centroid_weights<decltype(group_offsets)>{
      cumulative_weights->view().begin<double>(),
      group_offsets,
      tdigest_offsets.begin<size_type>()},
    centroid_group_info<decltype(group_offsets)>{cumulative_weights->view().begin<double>(),
                                                 group_offsets,
                                                 tdigest_offsets.begin<size_type>()},
    cumulative_centroid_weight<decltype(group_labels), decltype(group_offsets)>{
      cumulative_weights->view().begin<double>(),
      group_labels,
      group_offsets,
      {tdigest_offsets.begin<offset_type>(), static_cast<size_t>(tdigest_offsets.size())}},
    false,
    stream,
    mr);

  // input centroid values
  auto centroids = cudf::detail::make_counting_transform_iterator(
    0,
    make_weighted_centroid{merged->get_column(0).view().begin<double>(),
                           merged_weights.begin<double>()});

  // compute the tdigest
  return compute_tdigests(
    delta,
    centroids,
    centroids + merged->num_rows(),
    cumulative_centroid_weight<decltype(group_labels), decltype(group_offsets)>{
      cumulative_weights->view().begin<double>(),
      group_labels,
      group_offsets,
      {tdigest_offsets.begin<offset_type>(), static_cast<size_t>(tdigest_offsets.size())}},
    std::move(merged_min_col),
    std::move(merged_max_col),
    group_cluster_wl,
    std::move(group_cluster_offsets),
    total_clusters,
    false,
    stream,
    mr);
}

}  // anonymous namespace

std::unique_ptr<scalar> reduce_tdigest(column_view const& col,
                                       int max_centroids,
                                       rmm::cuda_stream_view stream,
                                       rmm::mr::device_memory_resource* mr)
{
  if (col.size() == 0) { return cudf::tdigest::detail::make_empty_tdigest_scalar(stream, mr); }

  // since this isn't coming out of a groupby, we need to sort the inputs in ascending
  // order with nulls at the end.
  table_view t({col});
  auto sorted = cudf::detail::sort(
    t, {order::ASCENDING}, {null_order::AFTER}, stream, rmm::mr::get_current_device_resource());

  auto const delta = max_centroids;
  return cudf::type_dispatcher(
    col.type(), typed_reduce_tdigest{}, sorted->get_column(0), delta, stream, mr);
}

std::unique_ptr<scalar> reduce_merge_tdigest(column_view const& input,
                                             int max_centroids,
                                             rmm::cuda_stream_view stream,
                                             rmm::mr::device_memory_resource* mr)
{
  tdigest_column_view tdv(input);

  if (input.size() == 0) { return cudf::tdigest::detail::make_empty_tdigest_scalar(stream, mr); }

  auto h_group_offsets = cudf::detail::make_counting_transform_iterator(
    0, [size = input.size()](size_type i) { return i == 0 ? 0 : size; });
  auto group_offsets = cudf::detail::make_counting_transform_iterator(
    0, [size = input.size()] __device__(size_type i) { return i == 0 ? 0 : size; });
  auto group_labels = thrust::make_constant_iterator(0);
  return to_tdigest_scalar(merge_tdigests(tdv,
                                          h_group_offsets,
                                          group_offsets,
                                          group_labels,
                                          input.size(),
                                          1,
                                          max_centroids,
                                          stream,
                                          mr),
                           stream,
                           mr);
}

std::unique_ptr<column> group_tdigest(column_view const& col,
                                      cudf::device_span<size_type const> group_offsets,
                                      cudf::device_span<size_type const> group_labels,
                                      cudf::device_span<size_type const> group_valid_counts,
                                      size_type num_groups,
                                      int max_centroids,
                                      rmm::cuda_stream_view stream,
                                      rmm::mr::device_memory_resource* mr)
{
  if (col.size() == 0) { return cudf::tdigest::detail::make_empty_tdigest_column(stream, mr); }

  auto const delta = max_centroids;
  return cudf::type_dispatcher(col.type(),
                               typed_group_tdigest{},
                               col,
                               group_offsets,
                               group_labels,
                               group_valid_counts,
                               num_groups,
                               delta,
                               stream,
                               mr);
}

std::unique_ptr<column> group_merge_tdigest(column_view const& input,
                                            cudf::device_span<size_type const> group_offsets,
                                            cudf::device_span<size_type const> group_labels,
                                            size_type num_groups,
                                            int max_centroids,
                                            rmm::cuda_stream_view stream,
                                            rmm::mr::device_memory_resource* mr)
{
  tdigest_column_view tdv(input);

  if (num_groups == 0 || input.size() == 0) {
    return cudf::tdigest::detail::make_empty_tdigest_column(stream, mr);
  }

  // bring group offsets back to the host
  std::vector<size_type> h_group_offsets(group_offsets.size());
  hipMemcpyAsync(h_group_offsets.data(),
                  group_offsets.begin(),
                  sizeof(size_type) * group_offsets.size(),
                  hipMemcpyDeviceToHost,
                  stream);

  return merge_tdigests(tdv,
                        h_group_offsets.begin(),
                        group_offsets.data(),
                        group_labels.data(),
                        group_labels.size(),
                        num_groups,
                        max_centroids,
                        stream,
                        mr);
}

}  // namespace detail
}  // namespace tdigest
}  // namespace cudf
