/*
 * Copyright (c) 2021-2022, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <vector>

#include <cudf/column/column_device_view.cuh>
#include <cudf/column/column_factories.hpp>
#include <cudf/copying.hpp>
#include <cudf/detail/iterator.cuh>
#include <cudf/detail/labeling/label_segments.cuh>
#include <cudf/detail/null_mask.hpp>
#include <cudf/detail/stream_compaction.hpp>
#include <cudf/detail/valid_if.cuh>
#include <cudf/lists/list_device_view.cuh>
#include <cudf/lists/lists_column_device_view.cuh>
#include <cudf/table/table.hpp>
#include <cudf/table/table_view.hpp>
#include <cudf/utilities/span.hpp>
#include <rmm/device_uvector.hpp>
#include <rmm/exec_policy.hpp>
#include <thrust/functional.h>
#include <thrust/logical.h>
#include <thrust/scan.h>
#include <thrust/tabulate.h>

#include "ColumnViewJni.hpp"

namespace cudf::jni {

std::unique_ptr<cudf::column>
new_column_with_boolean_column_as_validity(cudf::column_view const &exemplar,
                                           cudf::column_view const &validity_column) {
  CUDF_EXPECTS(validity_column.type().id() == type_id::BOOL8,
               "Validity column must be of type bool");
  CUDF_EXPECTS(validity_column.size() == exemplar.size(),
               "Exemplar and validity columns must have the same size");

  auto validity_device_view = cudf::column_device_view::create(validity_column);
  auto validity_begin = cudf::detail::make_optional_iterator<bool>(
      *validity_device_view, cudf::nullate::DYNAMIC{validity_column.has_nulls()});
  auto validity_end = validity_begin + validity_device_view->size();
  auto [null_mask, null_count] = cudf::detail::valid_if(
      validity_begin, validity_end,
      [] __device__(auto optional_bool) { return optional_bool.value_or(false); },
      cudf::get_default_stream());
  auto const exemplar_without_null_mask = cudf::column_view{
      exemplar.type(),
      exemplar.size(),
      exemplar.head<void>(),
      nullptr,
      0,
      exemplar.offset(),
      std::vector<cudf::column_view>{exemplar.child_begin(), exemplar.child_end()}};
  auto deep_copy = std::make_unique<cudf::column>(exemplar_without_null_mask);
  deep_copy->set_null_mask(std::move(null_mask), null_count);
  return deep_copy;
}

std::unique_ptr<cudf::column> generate_list_offsets(cudf::column_view const &list_length,
                                                    rmm::cuda_stream_view stream) {
  CUDF_EXPECTS(list_length.type().id() == cudf::type_id::INT32,
               "Input column does not have type INT32.");

  auto const begin_iter = list_length.template begin<cudf::size_type>();
  auto const end_iter = list_length.template end<cudf::size_type>();

  auto offsets_column = make_numeric_column(data_type{type_id::INT32}, list_length.size() + 1,
                                            mask_state::UNALLOCATED, stream);
  auto offsets_view = offsets_column->mutable_view();
  auto d_offsets = offsets_view.template begin<int32_t>();

  thrust::inclusive_scan(rmm::exec_policy(stream), begin_iter, end_iter, d_offsets + 1);
  CUDF_CUDA_TRY(hipMemsetAsync(d_offsets, 0, sizeof(int32_t), stream));

  return offsets_column;
}

namespace {

/**
 * @brief Check if the input list has any null elements.
 *
 * @param list The input list.
 * @return The boolean result indicating if the input list has null elements.
 */
__device__ bool list_has_nulls(list_device_view list) {
  return thrust::any_of(thrust::seq, thrust::make_counting_iterator(0),
                        thrust::make_counting_iterator(list.size()),
                        [&list](auto const idx) { return list.is_null(idx); });
}

} // namespace

void post_process_list_overlap(cudf::column_view const &lhs, cudf::column_view const &rhs,
                               std::unique_ptr<cudf::column> const &overlap_result,
                               rmm::cuda_stream_view stream) {
  // If both of the input columns do not have nulls, we don't need to do anything here.
  if (!lists_column_view{lhs}.child().has_nulls() && !lists_column_view{rhs}.child().has_nulls()) {
    return;
  }

  auto const overlap_cv = overlap_result->view();
  auto const lhs_cdv_ptr = column_device_view::create(lhs, stream);
  auto const rhs_cdv_ptr = column_device_view::create(rhs, stream);
  auto const overlap_cdv_ptr = column_device_view::create(overlap_cv, stream);

  // Create a new bitmask to satisfy Spark's arrays_overlap's special behavior.
  auto validity = rmm::device_uvector<bool>(overlap_cv.size(), stream);
  thrust::tabulate(rmm::exec_policy(stream), validity.begin(), validity.end(),
                   [lhs = cudf::detail::lists_column_device_view{*lhs_cdv_ptr},
                    rhs = cudf::detail::lists_column_device_view{*rhs_cdv_ptr},
                    overlap_result = *overlap_cdv_ptr] __device__(auto const idx) {
                     if (overlap_result.is_null(idx) ||
                         overlap_result.template element<bool>(idx)) {
                       return true;
                     }

                     // `lhs_list` and `rhs_list` should not be null, otherwise
                     // `overlap_result[idx]` is null and that has been handled above.
                     auto const lhs_list = list_device_view{lhs, idx};
                     auto const rhs_list = list_device_view{rhs, idx};

                     // Only proceed if both lists are non-empty.
                     if (lhs_list.size() == 0 || rhs_list.size() == 0) {
                       return true;
                     }

                     // Only proceed if at least one list has nulls.
                     if (!list_has_nulls(lhs_list) && !list_has_nulls(rhs_list)) {
                       return true;
                     }

                     // Here, the input lists satisfy all the conditions below so we output a
                     // null:
                     //  - Both of the the input lists have no non-null common element, and
                     //  - They are both non-empty, and
                     //  - Either of them contains null elements.
                     return false;
                   });

  // Create a new nullmask from the validity data.
  auto [new_null_mask, new_null_count] = cudf::detail::valid_if(
      validity.begin(), validity.end(), thrust::identity{}, cudf::get_default_stream());

  if (new_null_count > 0) {
    // If the `overlap_result` column is nullable, perform `bitmask_and` of its nullmask and the
    // new nullmask.
    if (overlap_cv.nullable()) {
      auto [null_mask, null_count] = cudf::detail::bitmask_and(
          std::vector<bitmask_type const *>{
              overlap_cv.null_mask(), static_cast<bitmask_type const *>(new_null_mask.data())},
          std::vector<cudf::size_type>{0, 0}, overlap_cv.size(), stream);
      overlap_result->set_null_mask(std::move(null_mask), null_count);
    } else {
      // Just set the output nullmask as the new nullmask.
      overlap_result->set_null_mask(std::move(new_null_mask), new_null_count);
    }
  }
}

std::unique_ptr<cudf::column> lists_distinct_by_key(cudf::lists_column_view const &input,
                                                    rmm::cuda_stream_view stream) {
  if (input.is_empty()) {
    return empty_like(input.parent());
  }

  auto const child = input.get_sliced_child(stream);

  // Generate labels for the input list elements.
  auto labels = rmm::device_uvector<cudf::size_type>(child.size(), stream);
  cudf::detail::label_segments(input.offsets_begin(), input.offsets_end(), labels.begin(),
                               labels.end(), stream);

  // Use `cudf::duplicate_keep_option::KEEP_LAST` so this will produce the desired behavior when
  // being called in `create_map` in spark-rapids.
  // Other options comparing nulls and NaNs are set as all-equal.
  auto out_columns = cudf::detail::stable_distinct(
                         table_view{{column_view{cudf::device_span<cudf::size_type const>{labels}},
                                     child.child(0), child.child(1)}}, // input table
                         std::vector<size_type>{0, 1},                 // key columns
                         cudf::duplicate_keep_option::KEEP_LAST, cudf::null_equality::EQUAL,
                         cudf::nan_equality::ALL_EQUAL, stream)
                         ->release();
  auto const out_labels = out_columns.front()->view();

  // Assemble a structs column of <out_keys, out_vals>.
  auto out_structs_members = std::vector<std::unique_ptr<cudf::column>>();
  out_structs_members.emplace_back(std::move(out_columns[1]));
  out_structs_members.emplace_back(std::move(out_columns[2]));
  auto out_structs =
      cudf::make_structs_column(out_labels.size(), std::move(out_structs_members), 0, {});

  // Assemble a lists column of structs<out_keys, out_vals>.
  auto out_offsets = make_numeric_column(data_type{type_to_id<offset_type>()}, input.size() + 1,
                                         mask_state::UNALLOCATED, stream);
  auto const offsets_begin = out_offsets->mutable_view().template begin<offset_type>();
  auto const labels_begin = out_labels.template begin<offset_type>();
  cudf::detail::labels_to_offsets(labels_begin, labels_begin + out_labels.size(), offsets_begin,
                                  offsets_begin + out_offsets->size(), stream);

  return cudf::make_lists_column(input.size(), std::move(out_offsets), std::move(out_structs),
                                 input.null_count(),
                                 cudf::detail::copy_bitmask(input.parent(), stream), stream);
}

} // namespace cudf::jni
