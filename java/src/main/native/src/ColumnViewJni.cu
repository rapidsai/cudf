/*
 * Copyright (c) 2021-2022, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <cudf/column/column_device_view.cuh>
#include <cudf/column/column_factories.hpp>
#include <cudf/copying.hpp>
#include <cudf/detail/iterator.cuh>
#include <cudf/detail/labeling/label_segments.cuh>
#include <cudf/detail/null_mask.hpp>
#include <cudf/detail/stream_compaction.hpp>
#include <cudf/detail/valid_if.cuh>
#include <cudf/table/table.hpp>
#include <cudf/table/table_view.hpp>
#include <cudf/utilities/span.hpp>
#include <rmm/device_uvector.hpp>
#include <rmm/exec_policy.hpp>
#include <thrust/scan.h>

#include "ColumnViewJni.hpp"

namespace cudf::jni {

std::unique_ptr<cudf::column>
new_column_with_boolean_column_as_validity(cudf::column_view const &exemplar,
                                           cudf::column_view const &validity_column) {
  CUDF_EXPECTS(validity_column.type().id() == type_id::BOOL8,
               "Validity column must be of type bool");
  CUDF_EXPECTS(validity_column.size() == exemplar.size(),
               "Exemplar and validity columns must have the same size");

  auto validity_device_view = cudf::column_device_view::create(validity_column);
  auto validity_begin = cudf::detail::make_optional_iterator<bool>(
      *validity_device_view, cudf::nullate::DYNAMIC{validity_column.has_nulls()});
  auto validity_end = validity_begin + validity_device_view->size();
  auto [null_mask, null_count] =
      cudf::detail::valid_if(validity_begin, validity_end, [] __device__(auto optional_bool) {
        return optional_bool.value_or(false);
      });
  auto const exemplar_without_null_mask = cudf::column_view{
      exemplar.type(),
      exemplar.size(),
      exemplar.head<void>(),
      nullptr,
      0,
      exemplar.offset(),
      std::vector<cudf::column_view>{exemplar.child_begin(), exemplar.child_end()}};
  auto deep_copy = std::make_unique<cudf::column>(exemplar_without_null_mask);
  deep_copy->set_null_mask(std::move(null_mask), null_count);
  return deep_copy;
}

std::unique_ptr<cudf::column> generate_list_offsets(cudf::column_view const &list_length,
                                                    rmm::cuda_stream_view stream) {
  CUDF_EXPECTS(list_length.type().id() == cudf::type_id::INT32,
               "Input column does not have type INT32.");

  auto const begin_iter = list_length.template begin<cudf::size_type>();
  auto const end_iter = list_length.template end<cudf::size_type>();

  auto offsets_column = make_numeric_column(data_type{type_id::INT32}, list_length.size() + 1,
                                            mask_state::UNALLOCATED, stream);
  auto offsets_view = offsets_column->mutable_view();
  auto d_offsets = offsets_view.template begin<int32_t>();

  thrust::inclusive_scan(rmm::exec_policy(stream), begin_iter, end_iter, d_offsets + 1);
  CUDF_CUDA_TRY(hipMemsetAsync(d_offsets, 0, sizeof(int32_t), stream));

  return offsets_column;
}

std::unique_ptr<cudf::column> lists_distinct_by_key(cudf::lists_column_view const &input,
                                                    rmm::cuda_stream_view stream) {
  if (input.is_empty()) {
    return empty_like(input.parent());
  }

  auto const child = input.get_sliced_child(stream);

  // Genereate labels for the input list elements.
  auto labels = rmm::device_uvector<cudf::size_type>(child.size(), stream);
  cudf::detail::label_segments(input.offsets_begin(), input.offsets_end(), labels.begin(),
                               labels.end(), stream);

  // Use `cudf::duplicate_keep_option::KEEP_LAST` so this will produce the desired behavior when
  // being called in `create_map` in spark-rapids.
  // Other options comparing nulls and NaNs are set as all-equal.
  auto out_columns = cudf::detail::stable_distinct(
                         table_view{{column_view{cudf::device_span<cudf::size_type const>{labels}},
                                     child.child(0), child.child(1)}}, // input table
                         std::vector<size_type>{0, 1},                 // key columns
                         cudf::duplicate_keep_option::KEEP_LAST, cudf::null_equality::EQUAL,
                         cudf::nan_equality::ALL_EQUAL, stream)
                         ->release();
  auto const out_labels = out_columns.front()->view();

  // Assemble a structs column of <out_keys, out_vals>.
  auto out_structs_members = std::vector<std::unique_ptr<cudf::column>>();
  out_structs_members.emplace_back(std::move(out_columns[1]));
  out_structs_members.emplace_back(std::move(out_columns[2]));
  auto out_structs =
      cudf::make_structs_column(out_labels.size(), std::move(out_structs_members), 0, {});

  // Assemble a lists column of structs<out_keys, out_vals>.
  auto out_offsets = make_numeric_column(data_type{type_to_id<offset_type>()}, input.size() + 1,
                                         mask_state::UNALLOCATED, stream);
  auto const offsets_begin = out_offsets->mutable_view().template begin<offset_type>();
  auto const labels_begin = out_labels.template begin<offset_type>();
  cudf::detail::labels_to_offsets(labels_begin, labels_begin + out_labels.size(), offsets_begin,
                                  offsets_begin + out_offsets->size(), stream);

  return cudf::make_lists_column(input.size(), std::move(out_offsets), std::move(out_structs),
                                 input.null_count(),
                                 cudf::detail::copy_bitmask(input.parent(), stream), stream);
}

} // namespace cudf::jni
