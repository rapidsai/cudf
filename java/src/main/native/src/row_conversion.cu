#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2020-2022, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <hip/hip_cooperative_groups.h>
#include <cudf/column/column_factories.hpp>
#include <cudf/detail/iterator.cuh>
#include <cudf/detail/sequence.hpp>
#include <cudf/detail/utilities/cuda.cuh>
#include <cudf/detail/utilities/integer_utils.hpp>
#include <cudf/detail/utilities/vector_factories.hpp>
#include <cudf/lists/lists_column_device_view.cuh>
#include <cudf/scalar/scalar_factories.hpp>
#include <cudf/strings/strings_column_view.hpp>
#include <cudf/table/table.hpp>
#include <cudf/types.hpp>
#include <cudf/utilities/bit.hpp>
#include <cudf/utilities/default_stream.hpp>
#include <cudf/utilities/error.hpp>
#include <cudf/utilities/traits.hpp>
#include <rmm/cuda_stream_view.hpp>
#include <rmm/device_buffer.hpp>
#include <rmm/device_uvector.hpp>
#include <rmm/exec_policy.hpp>
#include <thrust/binary_search.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/iterator/discard_iterator.h>
#include <thrust/iterator/transform_iterator.h>
#include <thrust/scan.h>
#include <type_traits>

#include "row_conversion.hpp"

#if defined(__CUDA_ARCH__) && __CUDA_ARCH__ >= 700
#define ASYNC_MEMCPY_SUPPORTED
#endif

#if !defined(__CUDA_ARCH__) || defined(ASYNC_MEMCPY_SUPPORTED)
#include <cuda/barrier>
#endif // #if !defined(__CUDA_ARCH__) || defined(ASYNC_MEMCPY_SUPPORTED)

#include <algorithm>
#include <cstdarg>
#include <cstdint>
#include <iostream>
#include <iterator>
#include <limits>
#include <optional>
#include <tuple>

namespace {

constexpr auto JCUDF_ROW_ALIGNMENT = 8;

constexpr auto MAX_BATCH_SIZE = std::numeric_limits<cudf::size_type>::max();

// Number of rows each block processes in the two kernels. Tuned via nsight
constexpr auto NUM_STRING_ROWS_PER_BLOCK_TO_ROWS = 1024;
constexpr auto NUM_STRING_ROWS_PER_BLOCK_FROM_ROWS = 64;
constexpr auto MIN_STRING_BLOCKS = 32;
constexpr auto MAX_STRING_BLOCKS = MAX_BATCH_SIZE;

constexpr auto NUM_WARPS_IN_BLOCK = 32;

} // anonymous namespace

// needed to suppress warning about cuda::barrier
#pragma nv_diag_suppress static_var_with_dynamic_init

using namespace cudf;
using detail::make_device_uvector_async;
using rmm::device_uvector;

#ifdef ASYNC_MEMCPY_SUPPORTED
using cuda::aligned_size_t;
#else
template <std::size_t> using aligned_size_t = size_t; // Local stub for cuda::aligned_size_t.
#endif // ASYNC_MEMCPY_SUPPORTED

namespace cudf {
namespace jni {
namespace detail {

/*
 * This module converts data from row-major to column-major and from column-major to row-major. It
 * is a transpose of the data of sorts, but there are a few complicating factors. They are spelled
 * out below:
 *
 * Row Batches:
 * The row data has to fit inside a cuDF column, which limits it to 2 gigs currently. The calling
 * code attempts to keep the data size under 2 gigs, but due to padding this isn't always the case,
 * so being able to break this up into multiple columns is necessary. Internally, this is referred
 * to as the row batch, which is a group of rows that will fit into this 2 gig space requirement.
 * There are typically 1 of these batches, but there can be 2.
 *
 * Async Memcpy:
 * The CUDA blocks are using memcpy_async, which allows for the device to schedule memcpy operations
 * and then wait on them to complete at a later time with a barrier. On Ampere or later hardware
 * there is dedicated hardware to do this copy and on pre-Ampere it should generate the same code
 * that a hand-rolled loop would generate, so performance should be the same or better than a
 * hand-rolled kernel.
 *
 * Tile Info:
 * Each CUDA block will work on a single tile info before exiting. This single tile consumes all
 * available shared memory. The kernel reads data into shared memory and then back out from shared
 * memory to device memory via memcpy_async. This kernel is completely memory bound.
 *
 * Batch Data:
 * This structure contains all the row batches and some book-keeping data necessary for the batches
 * such as row numbers for the batches.
 *
 * Tiles:
 * The tile info describes a tile of data to process. In a GPU with 48KB this equates to about 221
 * bytes in each direction of a table. The tiles are kept as square as possible to attempt to
 * coalesce memory operations. The taller a tile is the better coalescing of columns, but row
 * coalescing suffers. The wider a tile is the better the row coalescing, but columns coalescing
 * suffers. The code attempts to produce a square tile to balance the coalescing. It starts by
 * figuring out the optimal byte length and then adding columns to the data until the tile is too
 * large. Since rows are different width with different alignment requirements, this isn't typically
 * exact. Once a width is found the tiles are generated vertically with that width and height and
 * then the process repeats. This means all the tiles will be the same height, but will have
 * different widths based on what columns they encompass. Tiles in a vertical row will all have the
 * same dimensions.
 *
 *   --------------------------------
 *   | 4   5.0f || True   8   3   1 |
 *   | 3   6.0f || False  3   1   1 |
 *   | 2   7.0f || True   7   4   1 |
 *   | 1   8.0f || False  2   5   1 |
 *   --------------------------------
 *   | 0   9.0f || True   6   7   1 |
 *   ...
 */

/**
 * @brief The CUDA blocks work on one tile_info struct of data.
 *        This structure defines the workspaces for the blocks.
 *
 */
struct tile_info {
  int start_col;
  int start_row;
  int end_col;
  int end_row;
  int batch_number;

  __device__ inline size_type get_shared_row_size(size_type const *const col_offsets,
                                                  size_type const *const col_sizes) const {
    // this calculation is invalid if there are holes in the data such as a variable-width column.
    // It is wrong in a safe way in that it will say this row size is larger than it should be, so
    // we are not losing data we are just not as efficient as we could be with shared memory. This
    // may be a problem if the tile is computed without regard to variable width offset/length sizes
    // in that we overrun shared memory.
    return util::round_up_unsafe(col_offsets[end_col] + col_sizes[end_col] - col_offsets[start_col],
                                 JCUDF_ROW_ALIGNMENT);
  }

  __device__ inline size_type num_cols() const { return end_col - start_col + 1; }

  __device__ inline size_type num_rows() const { return end_row - start_row + 1; }
};

/**
 * @brief Returning rows is done in a byte cudf column. This is limited in size by
 *        `size_type` and so output is broken into batches of rows that fit inside
 *        this limit.
 *
 */
struct row_batch {
  size_type num_bytes;                     // number of bytes in this batch
  size_type row_count;                     // number of rows in the batch
  device_uvector<offset_type> row_offsets; // offsets column of output cudf column
};

/**
 * @brief Holds information about the batches of data to be processed
 *
 */
struct batch_data {
  device_uvector<size_type> batch_row_offsets;      // offsets to each row in incoming data
  device_uvector<size_type> d_batch_row_boundaries; // row numbers for the start of each batch
  std::vector<size_type>
      batch_row_boundaries;           // row numbers for the start of each batch: 0, 1500, 2700
  std::vector<row_batch> row_batches; // information about each batch such as byte count
};

/**
 * @brief builds row size information for tables that contain strings
 *
 * @param tbl table from which to compute row size information
 * @param fixed_width_and_validity_size size of fixed-width and validity data in this table
 * @param stream cuda stream on which to operate
 * @return pair of device vector of size_types of the row sizes of the table and a device vector of
 * offsets into the string column
 */
std::pair<rmm::device_uvector<size_type>, rmm::device_uvector<strings_column_view::offset_iterator>>
build_string_row_offsets(table_view const &tbl, size_type fixed_width_and_validity_size,
                         rmm::cuda_stream_view stream) {
  auto const num_rows = tbl.num_rows();
  rmm::device_uvector<size_type> d_row_sizes(num_rows, stream);
  thrust::uninitialized_fill(rmm::exec_policy(stream), d_row_sizes.begin(), d_row_sizes.end(), 0);

  auto d_offsets_iterators = [&]() {
    std::vector<strings_column_view::offset_iterator> offsets_iterators;
    auto offsets_iter = thrust::make_transform_iterator(
        tbl.begin(), [](auto const &col) -> strings_column_view::offset_iterator {
          if (!is_fixed_width(col.type())) {
            CUDF_EXPECTS(col.type().id() == type_id::STRING, "only string columns are supported!");
            return strings_column_view(col).offsets_begin();
          } else {
            return nullptr;
          }
        });
    std::copy_if(offsets_iter, offsets_iter + tbl.num_columns(),
                 std::back_inserter(offsets_iterators),
                 [](auto const &offset_ptr) { return offset_ptr != nullptr; });
    return make_device_uvector_async(offsets_iterators, stream);
  }();

  auto const num_columns = static_cast<size_type>(d_offsets_iterators.size());

  thrust::for_each(rmm::exec_policy(stream), thrust::make_counting_iterator(0),
                   thrust::make_counting_iterator(num_columns * num_rows),
                   [d_offsets_iterators = d_offsets_iterators.data(), num_columns, num_rows,
                    d_row_sizes = d_row_sizes.data()] __device__(auto element_idx) {
                     auto const row = element_idx % num_rows;
                     auto const col = element_idx / num_rows;
                     auto const val =
                         d_offsets_iterators[col][row + 1] - d_offsets_iterators[col][row];
                     atomicAdd(&d_row_sizes[row], val);
                   });

  // transform the row sizes to include fixed width size and alignment
  thrust::transform(rmm::exec_policy(stream), d_row_sizes.begin(), d_row_sizes.end(),
                    d_row_sizes.begin(), [fixed_width_and_validity_size] __device__(auto row_size) {
                      return util::round_up_unsafe(fixed_width_and_validity_size + row_size,
                                                   JCUDF_ROW_ALIGNMENT);
                    });

  return {std::move(d_row_sizes), std::move(d_offsets_iterators)};
}

/**
 * @brief functor to return the offset of a row in a table with string columns
 *
 */
struct string_row_offset_functor {
  string_row_offset_functor(device_span<size_type const> d_row_offsets)
      : d_row_offsets(d_row_offsets){};

  __device__ inline size_type operator()(int row_number, int) const {
    return d_row_offsets[row_number];
  }

  device_span<size_type const> d_row_offsets;
};

/**
 * @brief functor to return the offset of a row in a table with only fixed-width columns
 *
 */
struct fixed_width_row_offset_functor {
  fixed_width_row_offset_functor(size_type fixed_width_only_row_size)
      : _fixed_width_only_row_size(fixed_width_only_row_size){};

  __device__ inline size_type operator()(int row_number, int tile_row_start) const {
    return (row_number - tile_row_start) * _fixed_width_only_row_size;
  }

  size_type _fixed_width_only_row_size;
};

/**
 * @brief Copies data from row-based JCUDF format to column-based cudf format.
 *
 * This optimized version of the conversion is faster for fixed-width tables that do not have more
 * than 100 columns.
 *
 * @param num_rows number of rows in the incoming table
 * @param num_columns number of columns in the incoming table
 * @param row_size length in bytes of each row
 * @param input_offset_in_row offset to each row of data
 * @param num_bytes total number of bytes in the incoming data
 * @param output_data array of pointers to the output data
 * @param output_nm array of pointers to the output null masks
 * @param input_data pointing to the incoming row data
 */
__global__ void
copy_from_rows_fixed_width_optimized(const size_type num_rows, const size_type num_columns,
                                     const size_type row_size, const size_type *input_offset_in_row,
                                     const size_type *num_bytes, int8_t **output_data,
                                     bitmask_type **output_nm, const int8_t *input_data) {
  // We are going to copy the data in two passes.
  // The first pass copies a chunk of data into shared memory.
  // The second pass copies that chunk from shared memory out to the final location.

  // Because shared memory is limited we copy a subset of the rows at a time.
  // For simplicity we will refer to this as a row_group

  // In practice we have found writing more than 4 columns of data per thread
  // results in performance loss. As such we are using a 2 dimensional
  // kernel in terms of threads, but not in terms of blocks. Columns are
  // controlled by the y dimension (there is no y dimension in blocks). Rows
  // are controlled by the x dimension (there are multiple blocks in the x
  // dimension).

  size_type const rows_per_group = blockDim.x;
  size_type const row_group_start = blockIdx.x;
  size_type const row_group_stride = gridDim.x;
  size_type const row_group_end = (num_rows + rows_per_group - 1) / rows_per_group + 1;

  extern __shared__ int8_t shared_data[];

  // Because we are copying fixed width only data and we stride the rows
  // this thread will always start copying from shared data in the same place
  int8_t *row_tmp = &shared_data[row_size * threadIdx.x];
  int8_t *row_vld_tmp = &row_tmp[input_offset_in_row[num_columns - 1] + num_bytes[num_columns - 1]];

  for (auto row_group_index = row_group_start; row_group_index < row_group_end;
       row_group_index += row_group_stride) {
    // Step 1: Copy the data into shared memory
    // We know row_size is always aligned with and a multiple of int64_t;
    int64_t *long_shared = reinterpret_cast<int64_t *>(shared_data);
    int64_t const *long_input = reinterpret_cast<int64_t const *>(input_data);

    auto const shared_output_index = threadIdx.x + (threadIdx.y * blockDim.x);
    auto const shared_output_stride = blockDim.x * blockDim.y;
    auto const row_index_end = std::min(num_rows, ((row_group_index + 1) * rows_per_group));
    auto const num_rows_in_group = row_index_end - (row_group_index * rows_per_group);
    auto const shared_length = row_size * num_rows_in_group;

    size_type const shared_output_end = shared_length / sizeof(int64_t);

    auto const start_input_index = (row_size * row_group_index * rows_per_group) / sizeof(int64_t);

    for (size_type shared_index = shared_output_index; shared_index < shared_output_end;
         shared_index += shared_output_stride) {
      long_shared[shared_index] = long_input[start_input_index + shared_index];
    }
    // Wait for all of the data to be in shared memory
    __syncthreads();

    // Step 2 copy the data back out

    // Within the row group there should be 1 thread for each row.  This is a
    // requirement for launching the kernel
    auto const row_index = (row_group_index * rows_per_group) + threadIdx.x;
    // But we might not use all of the threads if the number of rows does not go
    // evenly into the thread count. We don't want those threads to exit yet
    // because we may need them to copy data in for the next row group.
    uint32_t active_mask = __ballot_sync(0xffff'ffffu, row_index < num_rows);
    if (row_index < num_rows) {
      auto const col_index_start = threadIdx.y;
      auto const col_index_stride = blockDim.y;
      for (auto col_index = col_index_start; col_index < num_columns;
           col_index += col_index_stride) {
        auto const col_size = num_bytes[col_index];
        int8_t const *col_tmp = &(row_tmp[input_offset_in_row[col_index]]);
        int8_t *col_output = output_data[col_index];
        switch (col_size) {
          case 1: {
            col_output[row_index] = *col_tmp;
            break;
          }
          case 2: {
            int16_t *short_col_output = reinterpret_cast<int16_t *>(col_output);
            short_col_output[row_index] = *reinterpret_cast<const int16_t *>(col_tmp);
            break;
          }
          case 4: {
            int32_t *int_col_output = reinterpret_cast<int32_t *>(col_output);
            int_col_output[row_index] = *reinterpret_cast<const int32_t *>(col_tmp);
            break;
          }
          case 8: {
            int64_t *long_col_output = reinterpret_cast<int64_t *>(col_output);
            long_col_output[row_index] = *reinterpret_cast<const int64_t *>(col_tmp);
            break;
          }
          default: {
            auto const output_offset = col_size * row_index;
            // TODO this should just not be supported for fixed width columns, but just in case...
            for (auto b = 0; b < col_size; b++) {
              col_output[b + output_offset] = col_tmp[b];
            }
            break;
          }
        }

        bitmask_type *nm = output_nm[col_index];
        int8_t *valid_byte = &row_vld_tmp[col_index / 8];
        size_type byte_bit_offset = col_index % 8;
        int predicate = *valid_byte & (1 << byte_bit_offset);
        uint32_t bitmask = __ballot_sync(active_mask, predicate);
        if (row_index % 32 == 0) {
          nm[word_index(row_index)] = bitmask;
        }
      } // end column loop
    }   // end row copy
    // wait for the row_group to be totally copied before starting on the next row group
    __syncthreads();
  }
}

__global__ void copy_to_rows_fixed_width_optimized(
    const size_type start_row, const size_type num_rows, const size_type num_columns,
    const size_type row_size, const size_type *output_offset_in_row, const size_type *num_bytes,
    const int8_t **input_data, const bitmask_type **input_nm, int8_t *output_data) {
  // We are going to copy the data in two passes.
  // The first pass copies a chunk of data into shared memory.
  // The second pass copies that chunk from shared memory out to the final location.

  // Because shared memory is limited we copy a subset of the rows at a time.
  // We do not support copying a subset of the columns in a row yet, so we don't
  // currently support a row that is wider than shared memory.
  // For simplicity we will refer to this as a row_group

  // In practice we have found reading more than 4 columns of data per thread
  // results in performance loss. As such we are using a 2 dimensional
  // kernel in terms of threads, but not in terms of blocks. Columns are
  // controlled by the y dimension (there is no y dimension in blocks). Rows
  // are controlled by the x dimension (there are multiple blocks in the x
  // dimension).

  size_type rows_per_group = blockDim.x;
  size_type row_group_start = blockIdx.x;
  size_type row_group_stride = gridDim.x;
  size_type row_group_end = (num_rows + rows_per_group - 1) / rows_per_group + 1;

  extern __shared__ int8_t shared_data[];

  // Because we are copying fixed width only data and we stride the rows
  // this thread will always start copying to shared data in the same place
  int8_t *row_tmp = &shared_data[row_size * threadIdx.x];
  int8_t *row_vld_tmp =
      &row_tmp[output_offset_in_row[num_columns - 1] + num_bytes[num_columns - 1]];

  for (size_type row_group_index = row_group_start; row_group_index < row_group_end;
       row_group_index += row_group_stride) {
    // Within the row group there should be 1 thread for each row.  This is a
    // requirement for launching the kernel
    size_type row_index = start_row + (row_group_index * rows_per_group) + threadIdx.x;
    // But we might not use all of the threads if the number of rows does not go
    // evenly into the thread count. We don't want those threads to exit yet
    // because we may need them to copy data back out.
    if (row_index < (start_row + num_rows)) {
      size_type col_index_start = threadIdx.y;
      size_type col_index_stride = blockDim.y;
      for (size_type col_index = col_index_start; col_index < num_columns;
           col_index += col_index_stride) {
        size_type col_size = num_bytes[col_index];
        int8_t *col_tmp = &(row_tmp[output_offset_in_row[col_index]]);
        const int8_t *col_input = input_data[col_index];
        switch (col_size) {
          case 1: {
            *col_tmp = col_input[row_index];
            break;
          }
          case 2: {
            const int16_t *short_col_input = reinterpret_cast<const int16_t *>(col_input);
            *reinterpret_cast<int16_t *>(col_tmp) = short_col_input[row_index];
            break;
          }
          case 4: {
            const int32_t *int_col_input = reinterpret_cast<const int32_t *>(col_input);
            *reinterpret_cast<int32_t *>(col_tmp) = int_col_input[row_index];
            break;
          }
          case 8: {
            const int64_t *long_col_input = reinterpret_cast<const int64_t *>(col_input);
            *reinterpret_cast<int64_t *>(col_tmp) = long_col_input[row_index];
            break;
          }
          default: {
            size_type input_offset = col_size * row_index;
            // TODO this should just not be supported for fixed width columns, but just in case...
            for (size_type b = 0; b < col_size; b++) {
              col_tmp[b] = col_input[b + input_offset];
            }
            break;
          }
        }
        // atomicOr only works on 32 bit or 64 bit  aligned values, and not byte aligned
        // so we have to rewrite the addresses to make sure that it is 4 byte aligned
        int8_t *valid_byte = &row_vld_tmp[col_index / 8];
        size_type byte_bit_offset = col_index % 8;
        uint64_t fixup_bytes = reinterpret_cast<uint64_t>(valid_byte) % 4;
        int32_t *valid_int = reinterpret_cast<int32_t *>(valid_byte - fixup_bytes);
        size_type int_bit_offset = byte_bit_offset + (fixup_bytes * 8);
        // Now copy validity for the column
        if (input_nm[col_index]) {
          if (bit_is_set(input_nm[col_index], row_index)) {
            atomicOr_block(valid_int, 1 << int_bit_offset);
          } else {
            atomicAnd_block(valid_int, ~(1 << int_bit_offset));
          }
        } else {
          // It is valid so just set the bit
          atomicOr_block(valid_int, 1 << int_bit_offset);
        }
      } // end column loop
    }   // end row copy
    // wait for the row_group to be totally copied into shared memory
    __syncthreads();

    // Step 2: Copy the data back out
    // We know row_size is always aligned with and a multiple of int64_t;
    int64_t *long_shared = reinterpret_cast<int64_t *>(shared_data);
    int64_t *long_output = reinterpret_cast<int64_t *>(output_data);

    size_type shared_input_index = threadIdx.x + (threadIdx.y * blockDim.x);
    size_type shared_input_stride = blockDim.x * blockDim.y;
    size_type row_index_end = ((row_group_index + 1) * rows_per_group);
    if (row_index_end > num_rows) {
      row_index_end = num_rows;
    }
    size_type num_rows_in_group = row_index_end - (row_group_index * rows_per_group);
    size_type shared_length = row_size * num_rows_in_group;

    size_type shared_input_end = shared_length / sizeof(int64_t);

    size_type start_output_index = (row_size * row_group_index * rows_per_group) / sizeof(int64_t);

    for (size_type shared_index = shared_input_index; shared_index < shared_input_end;
         shared_index += shared_input_stride) {
      long_output[start_output_index + shared_index] = long_shared[shared_index];
    }
    __syncthreads();
    // Go for the next round
  }
}

#ifdef ASYNC_MEMCPY_SUPPORTED
#define MEMCPY(dst, src, size, barrier) cuda::memcpy_async(dst, src, size, barrier)
#else
#define MEMCPY(dst, src, size, barrier) memcpy(dst, src, size)
#endif // ASYNC_MEMCPY_SUPPORTED

/**
 * @brief copy data from cudf columns into JCUDF format, which is row-based
 *
 * @tparam RowOffsetFunctor iterator that gives the size of a specific row of the table.
 * @param num_rows total number of rows in the table
 * @param num_columns total number of columns in the table
 * @param shmem_used_per_tile shared memory amount each `tile_info` is using
 * @param tile_infos span of `tile_info` structs the define the work
 * @param input_data pointer to raw table data
 * @param col_sizes array of sizes for each element in a column - one per column
 * @param col_offsets offset into input data row for each column's start
 * @param row_offsets offset to a specific row in the output data
 * @param batch_row_boundaries row numbers for batch starts
 * @param output_data pointer to output data
 *
 */
template <typename RowOffsetFunctor>
__global__ void copy_to_rows(const size_type num_rows, const size_type num_columns,
                             const size_type shmem_used_per_tile,
                             device_span<const tile_info> tile_infos, const int8_t **input_data,
                             const size_type *col_sizes, const size_type *col_offsets,
                             RowOffsetFunctor row_offsets, size_type const *batch_row_boundaries,
                             int8_t **output_data) {
  // We are going to copy the data in two passes.
  // The first pass copies a chunk of data into shared memory.
  // The second pass copies that chunk from shared memory out to the final location.

  // Because shared memory is limited we copy a subset of the rows at a time.
  // This has been broken up for us in the tile_info struct, so we don't have
  // any calculation to do here, but it is important to note.

  auto const group = cooperative_groups::this_thread_block();
  auto const warp = cooperative_groups::tiled_partition<cudf::detail::warp_size>(group);
  extern __shared__ int8_t shared_data[];

#ifdef ASYNC_MEMCPY_SUPPORTED
  __shared__ cuda::barrier<cuda::thread_scope_block> tile_barrier;
  if (group.thread_rank() == 0) {
    init(&tile_barrier, group.size());
  }
  group.sync();
#endif // ASYNC_MEMCPY_SUPPORTED

  auto const tile = tile_infos[blockIdx.x];
  auto const num_tile_cols = tile.num_cols();
  auto const num_tile_rows = tile.num_rows();
  auto const tile_row_size = tile.get_shared_row_size(col_offsets, col_sizes);
  auto const starting_column_offset = col_offsets[tile.start_col];

  // to do the copy we need to do n column copies followed by m element copies OR we have to do m
  // element copies followed by r row copies. When going from column to row it is much easier to
  // copy by elements first otherwise we would need a running total of the column sizes for our
  // tile, which isn't readily available. This makes it more appealing to copy element-wise from
  // input data into shared matching the end layout and do row-based memcopies out.

  // read each column across the tile
  // each warp takes a column with each thread of a warp taking a row this is done with cooperative
  // groups where each column is chosen by the tiled partition and each thread in that partition
  // works on a row
  for (int relative_col = warp.meta_group_rank(); relative_col < num_tile_cols;
       relative_col += warp.meta_group_size()) {

    auto const absolute_col = relative_col + tile.start_col;
    auto const col_size = col_sizes[absolute_col];
    auto const col_offset = col_offsets[absolute_col];
    auto const relative_col_offset = col_offset - starting_column_offset;
    auto const col_ptr = input_data[absolute_col];

    if (col_ptr == nullptr) {
      // variable-width data column
      continue;
    }

    for (int relative_row = warp.thread_rank(); relative_row < num_tile_rows;
         relative_row += warp.size()) {

      if (relative_row >= num_tile_rows) {
        // out of bounds
        continue;
      }
      auto const absolute_row = relative_row + tile.start_row;

      auto const shared_offset = relative_row * tile_row_size + relative_col_offset;
      auto const input_src = col_ptr + col_size * absolute_row;

      // copy the element from global memory
      switch (col_size) {
        case 2: {
          const int16_t *short_col_input = reinterpret_cast<const int16_t *>(input_src);
          *reinterpret_cast<int16_t *>(&shared_data[shared_offset]) = *short_col_input;
          break;
        }
        case 4: {
          const int32_t *int_col_input = reinterpret_cast<const int32_t *>(input_src);
          *reinterpret_cast<int32_t *>(&shared_data[shared_offset]) = *int_col_input;
          break;
        }
        case 8: {
          const int64_t *long_col_input = reinterpret_cast<const int64_t *>(input_src);
          *reinterpret_cast<int64_t *>(&shared_data[shared_offset]) = *long_col_input;
          break;
        }
        case 1: shared_data[shared_offset] = *input_src; break;
        default: {
          for (int i = 0; i < col_size; ++i) {
            shared_data[shared_offset] = *input_src;
          }
          break;
        }
      }
    }
  }

  auto const tile_output_buffer = output_data[tile.batch_number];
  auto const row_batch_start = tile.batch_number == 0 ? 0 : batch_row_boundaries[tile.batch_number];

  // no async copies above waiting on the barrier, so we sync the group here to ensure all copies to
  // shared memory are completed before copying data out
  group.sync();

  // each warp takes a row
  for (int copy_row = warp.meta_group_rank(); copy_row < tile.num_rows();
       copy_row += warp.meta_group_size()) {
    auto const src = &shared_data[tile_row_size * copy_row];
    auto const dst = tile_output_buffer + row_offsets(copy_row + tile.start_row, row_batch_start) +
                     starting_column_offset;
#ifdef ASYNC_MEMCPY_SUPPORTED
    cuda::memcpy_async(warp, dst, src, tile_row_size, tile_barrier);
#else
    for (int b = warp.thread_rank(); b < tile_row_size; b += warp.size()) {
      dst[b] = src[b];
    }
#endif
  }

#ifdef ASYNC_MEMCPY_SUPPORTED
  // wait on the last copies to complete
  tile_barrier.arrive_and_wait();
#else
  group.sync();
#endif // ASYNC_MEMCPY_SUPPORTED
}

/**
 * @brief copy data from row-based format to cudf columns
 *
 * @tparam RowOffsetFunctor iterator that gives the size of a specific row of the table.
 * @param num_rows total number of rows in the table
 * @param num_columns total number of columns in the table
 * @param shmem_used_per_tile amount of shared memory that is used by a tile
 * @param row_offsets offset to a specific row in the output data
 * @param batch_row_boundaries row numbers for batch starts
 * @param output_data pointer to output data, partitioned by data size
 * @param validity_offsets offset into input data row for validity data
 * @param tile_infos information about the tiles of work
 * @param input_nm pointer to input data
 *
 */
template <typename RowOffsetFunctor>
__global__ void
copy_validity_to_rows(const size_type num_rows, const size_type num_columns,
                      const size_type shmem_used_per_tile, RowOffsetFunctor row_offsets,
                      size_type const *batch_row_boundaries, int8_t **output_data,
                      const size_type validity_offset, device_span<const tile_info> tile_infos,
                      const bitmask_type **input_nm) {
  extern __shared__ int8_t shared_data[];

  // each thread of warp reads a single int32 of validity - so we read 128 bytes then ballot_sync
  // the bits and write the result to shmem after we fill shared mem memcpy it out in a blob.
  auto const group = cooperative_groups::this_thread_block();
  auto const warp = cooperative_groups::tiled_partition<cudf::detail::warp_size>(group);

#ifdef ASYNC_MEMCPY_SUPPORTED
  // Initialize cuda barriers for each tile.
  __shared__ cuda::barrier<cuda::thread_scope_block> shared_tile_barrier;
  if (group.thread_rank() == 0) {
    init(&shared_tile_barrier, group.size());
  }
  group.sync();
#endif // ASYNC_MEMCPY_SUPPORTED

  auto tile = tile_infos[blockIdx.x];
  auto const num_tile_cols = tile.num_cols();
  auto const num_tile_rows = tile.num_rows();

  auto const threads_per_warp = warp.size();
  auto const rows_per_read = cudf::detail::size_in_bits<bitmask_type>();

  auto const num_sections_x = util::div_rounding_up_unsafe(num_tile_cols, threads_per_warp);
  auto const num_sections_y = util::div_rounding_up_unsafe(num_tile_rows, rows_per_read);
  auto const validity_data_row_length = util::round_up_unsafe(
      util::div_rounding_up_unsafe(num_tile_cols, CHAR_BIT), JCUDF_ROW_ALIGNMENT);
  auto const total_sections = num_sections_x * num_sections_y;

  // the tile is divided into sections. A warp operates on a section at a time.
  for (int my_section_idx = warp.meta_group_rank(); my_section_idx < total_sections;
       my_section_idx += warp.meta_group_size()) {
    // convert to rows and cols
    auto const section_x = my_section_idx % num_sections_x;
    auto const section_y = my_section_idx / num_sections_x;
    auto const relative_col = section_x * threads_per_warp + warp.thread_rank();
    auto const relative_row = section_y * rows_per_read;
    auto const absolute_col = relative_col + tile.start_col;
    auto const absolute_row = relative_row + tile.start_row;
    auto const participating = absolute_col < num_columns && absolute_row < num_rows;
    auto const participation_mask = __ballot_sync(0xFFFF'FFFFu, participating);

    if (participating) {
      auto my_data = input_nm[absolute_col] != nullptr ?
                         input_nm[absolute_col][word_index(absolute_row)] :
                         std::numeric_limits<uint32_t>::max();

      // every thread that is participating in the warp has 4 bytes, but it's column-based data and
      // we need it in row-based. So we shuffle the bits around with ballot_sync to make the bytes
      // we actually write.
      bitmask_type dw_mask = 0x1;
      for (int i = 0; i < threads_per_warp && relative_row + i < num_rows; ++i, dw_mask <<= 1) {
        auto validity_data = __ballot_sync(participation_mask, my_data & dw_mask);
        // lead thread in each warp writes data
        auto const validity_write_offset =
            validity_data_row_length * (relative_row + i) + (relative_col / CHAR_BIT);
        if (warp.thread_rank() == 0) {
          *reinterpret_cast<bitmask_type *>(&shared_data[validity_write_offset]) = validity_data;
        }
      }
    }
  }

  auto const output_data_base =
      output_data[tile.batch_number] + validity_offset + tile.start_col / CHAR_BIT;

  // each warp copies a row at a time
  auto const row_bytes = util::div_rounding_up_unsafe(num_tile_cols, CHAR_BIT);
  auto const row_batch_start = tile.batch_number == 0 ? 0 : batch_row_boundaries[tile.batch_number];

  // make sure entire tile has finished copy
  // Note that this was copied from above just under the for loop due to nsight complaints about
  // divergent threads
  group.sync();

  for (int relative_row = warp.meta_group_rank(); relative_row < num_tile_rows;
       relative_row += warp.meta_group_size()) {
    auto const src = &shared_data[validity_data_row_length * relative_row];
    auto const dst = output_data_base + row_offsets(relative_row + tile.start_row, row_batch_start);
#ifdef ASYNC_MEMCPY_SUPPORTED
    cuda::memcpy_async(warp, dst, src, row_bytes, shared_tile_barrier);
#else
    for (int b = warp.thread_rank(); b < row_bytes; b += warp.size()) {
      dst[b] = src[b];
    }
#endif
  }

#ifdef ASYNC_MEMCPY_SUPPORTED
  // wait for tile of data to arrive
  shared_tile_barrier.arrive_and_wait();
#else
  group.sync();
#endif // ASYNC_MEMCPY_SUPPORTED
}

/**
 * @brief kernel to copy string data to JCUDF row format
 *
 * @tparam RowOffsetFunctor iterator for row offsets into the destination data
 * @param num_rows number of rows in this portion of the table
 * @param num_variable_columns number of columns of variable-width data
 * @param variable_input_data variable width data column pointers
 * @param variable_col_output_offsets output offset information for variable-width columns
 * @param variable_col_offsets input offset information for variable-width columns
 * @param fixed_width_row_size offset to variable-width data in a row
 * @param row_offsets offsets for each row in output data
 * @param batch_row_offset row start for this batch
 * @param output_data pointer to output data for this batch
 *
 */
template <typename RowOffsetFunctor>
__global__ void copy_strings_to_rows(size_type const num_rows, size_type const num_variable_columns,
                                     int8_t const **variable_input_data,
                                     size_type const *variable_col_output_offsets,
                                     size_type const **variable_col_offsets,
                                     size_type fixed_width_row_size, RowOffsetFunctor row_offsets,
                                     size_type const batch_row_offset, int8_t *output_data) {
  // Each block will take a group of rows controlled by NUM_STRING_ROWS_PER_BLOCK_TO_ROWS. Each warp
  // will copy a row at a time. The base thread will first go through column data and fill out
  // offset/length information for the column. Then all threads of the warp will participate in the
  // memcpy of the string data.
  auto const my_block = cooperative_groups::this_thread_block();
  auto const warp = cooperative_groups::tiled_partition<cudf::detail::warp_size>(my_block);
#ifdef ASYNC_MEMCPY_SUPPORTED
  cuda::barrier<cuda::thread_scope_block> block_barrier;
#endif

  auto const start_row =
      blockIdx.x * NUM_STRING_ROWS_PER_BLOCK_TO_ROWS + warp.meta_group_rank() + batch_row_offset;
  auto const end_row =
      std::min(num_rows, static_cast<size_type>(start_row + NUM_STRING_ROWS_PER_BLOCK_TO_ROWS));

  for (int row = start_row; row < end_row; row += warp.meta_group_size()) {
    auto offset = fixed_width_row_size; // initial offset to variable-width data
    auto const base_row_offset = row_offsets(row, 0);
    for (int col = 0; col < num_variable_columns; ++col) {
      auto const string_start_offset = variable_col_offsets[col][row];
      auto const string_length = variable_col_offsets[col][row + 1] - string_start_offset;
      if (warp.thread_rank() == 0) {
        // write the offset/length to column
        uint32_t *output_dest = reinterpret_cast<uint32_t *>(
            &output_data[base_row_offset + variable_col_output_offsets[col]]);
        output_dest[0] = offset;
        output_dest[1] = string_length;
      }
      auto string_output_dest = &output_data[base_row_offset + offset];
      auto string_output_src = &variable_input_data[col][string_start_offset];
      warp.sync();
#ifdef ASYNC_MEMCPY_SUPPORTED
      cuda::memcpy_async(warp, string_output_dest, string_output_src, string_length, block_barrier);
#else
      for (int c = warp.thread_rank(); c < string_length; c += warp.size()) {
        string_output_dest[c] = string_output_src[c];
      }
#endif
      offset += string_length;
    }
  }
}
/**
 * @brief copy data from row-based format to cudf columns
 *
 * @tparam RowOffsetFunctor iterator that gives the size of a specific row of the table.
 * @param num_rows total number of rows in the table
 * @param num_columns total number of columns in the table
 * @param shmem_used_per_tile amount of shared memory that is used by a tile
 * @param row_offsets offset to a specific row in the input data
 * @param batch_row_boundaries row numbers for batch starts
 * @param output_data pointers to column data
 * @param col_sizes array of sizes for each element in a column - one per column
 * @param col_offsets offset into input data row for each column's start
 * @param tile_infos information about the tiles of work
 * @param input_data pointer to input data
 *
 */
template <typename RowOffsetFunctor>
__global__ void copy_from_rows(const size_type num_rows, const size_type num_columns,
                               const size_type shmem_used_per_tile, RowOffsetFunctor row_offsets,
                               size_type const *batch_row_boundaries, int8_t **output_data,
                               const size_type *col_sizes, const size_type *col_offsets,
                               device_span<const tile_info> tile_infos, const int8_t *input_data) {
  // We are going to copy the data in two passes.
  // The first pass copies a chunk of data into shared memory.
  // The second pass copies that chunk from shared memory out to the final location.

  // Because shared memory is limited we copy a subset of the rows at a time. This has been broken
  // up for us in the tile_info struct, so we don't have any calculation to do here, but it is
  // important to note.

  // To speed up some of the random access memory we do, we copy col_sizes and col_offsets to shared
  // memory for each of the tiles that we work on

  auto const group = cooperative_groups::this_thread_block();
  auto const warp = cooperative_groups::tiled_partition<cudf::detail::warp_size>(group);
  extern __shared__ int8_t shared[];

#ifdef ASYNC_MEMCPY_SUPPORTED
  // Initialize cuda barriers for each tile.
  __shared__ cuda::barrier<cuda::thread_scope_block> tile_barrier;
  if (group.thread_rank() == 0) {
    init(&tile_barrier, group.size());
  }
  group.sync();
#endif // ASYNC_MEMCPY_SUPPORTED

  {
    auto const fetch_tile = tile_infos[blockIdx.x];
    auto const fetch_tile_start_row = fetch_tile.start_row;
    auto const starting_col_offset = col_offsets[fetch_tile.start_col];
    auto const fetch_tile_row_size = fetch_tile.get_shared_row_size(col_offsets, col_sizes);
    auto const row_batch_start =
        fetch_tile.batch_number == 0 ? 0 : batch_row_boundaries[fetch_tile.batch_number];

    for (int absolute_row = warp.meta_group_rank() + fetch_tile.start_row;
         absolute_row <= fetch_tile.end_row; absolute_row += warp.meta_group_size()) {
      warp.sync();
      auto shared_offset = (absolute_row - fetch_tile_start_row) * fetch_tile_row_size;
      auto dst = &shared[shared_offset];
      auto src = &input_data[row_offsets(absolute_row, row_batch_start) + starting_col_offset];
      // copy the data
#ifdef ASYNC_MEMCPY_SUPPORTED
      cuda::memcpy_async(warp, dst, src, fetch_tile_row_size, tile_barrier);
#else
      for (int b = warp.thread_rank(); b < fetch_tile_row_size; b += warp.size()) {
        dst[b] = src[b];
      }
#endif
    }
  }

  {
    auto const tile = tile_infos[blockIdx.x];
    auto const rows_in_tile = tile.num_rows();
    auto const cols_in_tile = tile.num_cols();
    auto const tile_row_size = tile.get_shared_row_size(col_offsets, col_sizes);

#ifdef ASYNC_MEMCPY_SUPPORTED
    // ensure our data is ready
    tile_barrier.arrive_and_wait();
#else
    group.sync();
#endif // ASYNC_MEMCPY_SUPPORTED

    // Now we copy from shared memory to final destination. The data is laid out in rows in shared
    // memory, so the reads for a column will be "vertical". Because of this and the different sizes
    // for each column, this portion is handled on row/column basis. to prevent each thread working
    // on a single row and also to ensure that all threads can do work in the case of more threads
    // than rows, we do a global index instead of a double for loop with col/row.
    for (int relative_row = warp.thread_rank(); relative_row < rows_in_tile;
         relative_row += warp.size()) {

      auto const absolute_row = relative_row + tile.start_row;
      auto const shared_memory_row_offset = tile_row_size * relative_row;

      for (int relative_col = warp.meta_group_rank(); relative_col < cols_in_tile;
           relative_col += warp.meta_group_size()) {
        auto const absolute_col = relative_col + tile.start_col;

        auto const shared_memory_offset =
            col_offsets[absolute_col] - col_offsets[tile.start_col] + shared_memory_row_offset;
        auto const column_size = col_sizes[absolute_col];

        int8_t *shmem_src = &shared[shared_memory_offset];
        int8_t *dst = &output_data[absolute_col][absolute_row * column_size];

        MEMCPY(dst, shmem_src, column_size, tile_barrier);
      }
    }
  }

#ifdef ASYNC_MEMCPY_SUPPORTED
  // wait on the last copies to complete
  tile_barrier.arrive_and_wait();
#else
  group.sync();
#endif // ASYNC_MEMCPY_SUPPORTED
}

/**
 * @brief copy data from row-based format to cudf columns
 *
 * @tparam RowOffsetFunctor iterator that gives the size of a specific row of the table.
 * @param num_rows total number of rows in the table
 * @param num_columns total number of columns in the table
 * @param shmem_used_per_tile amount of shared memory that is used by a tile
 * @param row_offsets offset to the first column a specific row in the input data
 * @param batch_row_boundaries row numbers for batch starts
 * @param output_nm pointers to null masks for columns
 * @param validity_offsets offset into input data row for validity data
 * @param tile_infos information about the tiles of work
 * @param input_data pointer to input data
 *
 */
template <typename RowOffsetFunctor>
__global__ void
copy_validity_from_rows(const size_type num_rows, const size_type num_columns,
                        const size_type shmem_used_per_tile, RowOffsetFunctor row_offsets,
                        size_type const *batch_row_boundaries, bitmask_type **output_nm,
                        const size_type validity_offset, device_span<const tile_info> tile_infos,
                        const int8_t *input_data) {
  extern __shared__ int8_t shared[];

  using cudf::detail::warp_size;

  // each thread of warp reads a single byte of validity - so we read 32 bytes then ballot_sync the
  // bits and write the result to shmem after we fill shared mem memcpy it out in a blob. Probably
  // need knobs for number of rows vs columns to balance read/write

  //        C0  C1  C2  C3  C4  C5  C6  C7
  //  R0    1   0   1   0   0   1   1   0       <-- thread 0 reads byte r0
  //  R1    1   1   1   1   1   1   1   0       <-- thread 1 reads byte r1
  //  R2    0   0   1   0   0   1   1   0       <-- thread 2 reads byte r2
  //  ...
  //  R31   1   1   1   1   1   1   1   1       <-- thread 31 reads byte r31
  //        ^
  //        |  1 bit of each input byte, by column, are swizzled into a single 32 bit word via
  //        __ballot_sync, representing 32 rows of that column.

  auto const group = cooperative_groups::this_thread_block();
  auto const warp = cooperative_groups::tiled_partition<cudf::detail::warp_size>(group);

#ifdef ASYNC_MEMCPY_SUPPORTED
  // Initialize cuda barriers for each tile.
  __shared__ cuda::barrier<cuda::thread_scope_block> shared_tile_barrier;
  if (group.thread_rank() == 0) {
    init(&shared_tile_barrier, group.size());
  }
  group.sync();
#endif // ASYNC_MEMCPY_SUPPORTED

  auto const tile = tile_infos[blockIdx.x];
  auto const tile_start_col = tile.start_col;
  auto const tile_start_row = tile.start_row;
  auto const num_tile_cols = tile.num_cols();
  auto const num_tile_rows = tile.num_rows();

  auto const threads_per_warp = warp.size();
  auto const cols_per_read = CHAR_BIT;

  auto const rows_per_read = static_cast<size_type>(threads_per_warp);
  auto const num_sections_x = util::div_rounding_up_safe(num_tile_cols, cols_per_read);
  auto const num_sections_y = util::div_rounding_up_safe(num_tile_rows, rows_per_read);
  auto const validity_data_col_length = num_sections_y * 4; // words to bytes
  auto const total_sections = num_sections_x * num_sections_y;

  // the tile is divided into sections. A warp operates on a section at a time.
  for (int my_section_idx = warp.meta_group_rank(); my_section_idx < total_sections;
       my_section_idx += warp.meta_group_size()) {
    // convert section to row and col
    auto const section_x = my_section_idx % num_sections_x;
    auto const section_y = my_section_idx / num_sections_x;
    auto const relative_col = section_x * cols_per_read;
    auto const relative_row = section_y * rows_per_read + warp.thread_rank();
    auto const absolute_col = relative_col + tile_start_col;
    auto const absolute_row = relative_row + tile_start_row;
    auto const row_batch_start =
        tile.batch_number == 0 ? 0 : batch_row_boundaries[tile.batch_number];

    auto const participation_mask = __ballot_sync(0xFFFF'FFFFu, absolute_row < num_rows);

    if (absolute_row < num_rows) {
      auto const my_byte = input_data[row_offsets(absolute_row, row_batch_start) + validity_offset +
                                      (absolute_col / cols_per_read)];

      // so every thread that is participating in the warp has a byte, but it's row-based data and
      // we need it in column-based. So we shuffle the bits around to make the bytes we actually
      // write.
      for (int i = 0, byte_mask = 0x1; (i < cols_per_read) && ((relative_col + i) < num_columns);
           ++i, byte_mask <<= 1) {
        auto const validity_data = __ballot_sync(participation_mask, my_byte & byte_mask);
        // lead thread in each warp writes data
        if (warp.thread_rank() == 0) {
          auto const validity_write_offset =
              validity_data_col_length * (relative_col + i) + relative_row / cols_per_read;
          *reinterpret_cast<bitmask_type *>(&shared[validity_write_offset]) = validity_data;
        }
      }
    }
  }

  // now memcpy the shared memory out to the final destination
  auto const col_words = util::div_rounding_up_unsafe(num_tile_rows, CHAR_BIT * 4);

  // make sure entire tile has finished copy
  group.sync();

  for (int relative_col = warp.meta_group_rank(); relative_col < num_tile_cols;
       relative_col += warp.meta_group_size()) {
    auto const absolute_col = relative_col + tile_start_col;
    auto dst = output_nm[absolute_col] + word_index(tile_start_row);
    auto const src =
        reinterpret_cast<bitmask_type *>(&shared[validity_data_col_length * relative_col]);

#ifdef ASYNC_MEMCPY_SUPPORTED
    cuda::memcpy_async(warp, dst, src, aligned_size_t<4>(validity_data_col_length),
                       shared_tile_barrier);
#else
    for (int b = warp.thread_rank(); b < col_words; b += warp.size()) {
      dst[b] = src[b];
    }
#endif
  }

#ifdef ASYNC_MEMCPY_SUPPORTED
  // wait for tile of data to arrive
  shared_tile_barrier.arrive_and_wait();
#else
  group.sync();
#endif // ASYNC_MEMCPY_SUPPORTED
}

/**
 * @brief copies string data from jcudf row format to cudf columns
 *
 * @tparam RowOffsetFunctor iterator for row offsets into the destination data
 * @param row_offsets offsets for each row in input data
 * @param string_row_offsets offset data into jcudf row data for each string
 * @param string_lengths length of each incoming string in each column
 * @param string_column_offsets offset column data for cudf column
 * @param string_col_data output cudf string column data
 * @param row_data jcudf row data
 * @param num_rows number of rows in data
 * @param num_string_columns number of string columns in the table
 */
template <typename RowOffsetFunctor>
__global__ void copy_strings_from_rows(RowOffsetFunctor row_offsets, int32_t **string_row_offsets,
                                       int32_t **string_lengths, size_type **string_column_offsets,
                                       char **string_col_data, int8_t const *row_data,
                                       size_type const num_rows,
                                       size_type const num_string_columns) {
  // Each warp takes a tile, which is a single column and up to ROWS_PER_BLOCK rows. A tile will not
  // wrap around the bottom of the table. The warp will copy the strings for each row in the tile.
  // Traversing in row-major order to coalesce the offsets and size reads.
  auto my_block = cooperative_groups::this_thread_block();
  auto warp = cooperative_groups::tiled_partition<cudf::detail::warp_size>(my_block);
#ifdef ASYNC_MEMCPY_SUPPORTED
  cuda::barrier<cuda::thread_scope_block> block_barrier;
#endif

  // workaround for not being able to take a reference to a constexpr host variable
  auto const ROWS_PER_BLOCK = NUM_STRING_ROWS_PER_BLOCK_FROM_ROWS;
  auto const tiles_per_col = util::div_rounding_up_unsafe(num_rows, ROWS_PER_BLOCK);
  auto const starting_tile = blockIdx.x * warp.meta_group_size() + warp.meta_group_rank();
  auto const num_tiles = tiles_per_col * num_string_columns;
  auto const tile_stride = warp.meta_group_size() * gridDim.x;
  // Each warp will copy strings in its tile. This is handled by all the threads of a warp passing
  // the same parameters to async_memcpy and all threads in the warp participating in the copy.
  for (auto my_tile = starting_tile; my_tile < num_tiles; my_tile += tile_stride) {
    auto const starting_row = (my_tile % tiles_per_col) * ROWS_PER_BLOCK;
    auto const col = my_tile / tiles_per_col;
    auto const str_len = string_lengths[col];
    auto const str_row_off = string_row_offsets[col];
    auto const str_col_off = string_column_offsets[col];
    auto str_col_data = string_col_data[col];
    for (int row = starting_row; row < starting_row + ROWS_PER_BLOCK && row < num_rows; ++row) {
      auto const src = &row_data[row_offsets(row, 0) + str_row_off[row]];
      auto dst = &str_col_data[str_col_off[row]];

#ifdef ASYNC_MEMCPY_SUPPORTED
      cuda::memcpy_async(warp, dst, src, str_len[row], block_barrier);
#else
      for (int c = warp.thread_rank(); c < str_len[row]; c += warp.size()) {
        dst[c] = src[c];
      }
#endif
    }
  }
}

/**
 * @brief Calculate the dimensions of the kernel for fixed width only columns.
 *
 * @param [in] num_columns the number of columns being copied.
 * @param [in] num_rows the number of rows being copied.
 * @param [in] size_per_row the size each row takes up when padded.
 * @param [out] blocks the size of the blocks for the kernel
 * @param [out] threads the size of the threads for the kernel
 * @return the size in bytes of shared memory needed for each block.
 */
static int calc_fixed_width_kernel_dims(const size_type num_columns, const size_type num_rows,
                                        const size_type size_per_row, dim3 &blocks, dim3 &threads) {
  // We have found speed degrades when a thread handles more than 4 columns.
  // Each block is 2 dimensional. The y dimension indicates the columns.
  // We limit this to 32 threads in the y dimension so we can still
  // have at least 32 threads in the x dimension (1 warp) which should
  // result in better coalescing of memory operations. We also
  // want to guarantee that we are processing a multiple of 32 threads
  // in the x dimension because we use atomic operations at the block
  // level when writing validity data out to main memory, and that would
  // need to change if we split a word of validity data between blocks.
  int const y_block_size = min(util::div_rounding_up_safe(num_columns, 4), 32);
  int const x_possible_block_size = 1024 / y_block_size;
  // 48KB is the default setting for shared memory per block according to the cuda tutorials
  // If someone configures the GPU to only have 16 KB this might not work.
  int const max_shared_size = 48 * 1024;
  // If we don't have enough shared memory there is no point in having more threads
  // per block that will just sit idle
  auto const max_block_size = std::min(x_possible_block_size, max_shared_size / size_per_row);
  // Make sure that the x dimension is a multiple of 32 this not only helps
  // coalesce memory access it also lets us do a ballot sync for validity to write
  // the data back out the warp level.  If x is a multiple of 32 then each thread in the y
  // dimension is associated with one or more warps, that should correspond to the validity
  // words directly.
  int const block_size = (max_block_size / 32) * 32;
  CUDF_EXPECTS(block_size != 0, "Row size is too large to fit in shared memory");

  // The maximum number of blocks supported in the x dimension is 2 ^ 31 - 1
  // but in practice having too many can cause some overhead that I don't totally
  // understand. Playing around with this having as little as 600 blocks appears
  // to be able to saturate memory on V100, so this is an order of magnitude higher
  // to try and future proof this a bit.
  int const num_blocks = std::clamp((num_rows + block_size - 1) / block_size, 1, 10240);

  blocks.x = num_blocks;
  blocks.y = 1;
  blocks.z = 1;
  threads.x = block_size;
  threads.y = y_block_size;
  threads.z = 1;
  return size_per_row * block_size;
}

/**
 * When converting to rows it is possible that the size of the table was too big to fit
 * in a single column. This creates an output column for a subset of the rows in a table
 * going from start row and containing the next num_rows.  Most of the parameters passed
 * into this function are common between runs and should be calculated once.
 */
static std::unique_ptr<column> fixed_width_convert_to_rows(
    const size_type start_row, const size_type num_rows, const size_type num_columns,
    const size_type size_per_row, rmm::device_uvector<size_type> &column_start,
    rmm::device_uvector<size_type> &column_size, rmm::device_uvector<const int8_t *> &input_data,
    rmm::device_uvector<const bitmask_type *> &input_nm, const scalar &zero,
    const scalar &scalar_size_per_row, rmm::cuda_stream_view stream,
    rmm::mr::device_memory_resource *mr) {
  int64_t const total_allocation = size_per_row * num_rows;
  // We made a mistake in the split somehow
  CUDF_EXPECTS(total_allocation < std::numeric_limits<size_type>::max(),
               "Table is too large to fit!");

  // Allocate and set the offsets row for the byte array
  std::unique_ptr<column> offsets =
      cudf::detail::sequence(num_rows + 1, zero, scalar_size_per_row, stream);

  std::unique_ptr<column> data =
      make_numeric_column(data_type(type_id::INT8), static_cast<size_type>(total_allocation),
                          mask_state::UNALLOCATED, stream, mr);

  dim3 blocks;
  dim3 threads;
  int shared_size =
      detail::calc_fixed_width_kernel_dims(num_columns, num_rows, size_per_row, blocks, threads);

  copy_to_rows_fixed_width_optimized<<<blocks, threads, shared_size, stream.value()>>>(
      start_row, num_rows, num_columns, size_per_row, column_start.data(), column_size.data(),
      input_data.data(), input_nm.data(), data->mutable_view().data<int8_t>());

  return make_lists_column(num_rows, std::move(offsets), std::move(data), 0,
                           rmm::device_buffer{0, stream, mr}, stream, mr);
}

static inline bool are_all_fixed_width(std::vector<data_type> const &schema) {
  return std::all_of(schema.begin(), schema.end(),
                     [](const data_type &t) { return is_fixed_width(t); });
}

/**
 * @brief Given a set of fixed width columns, calculate how the data will be laid out in memory.
 *
 * @param [in] schema the types of columns that need to be laid out.
 * @param [out] column_start the byte offset where each column starts in the row.
 * @param [out] column_size the size in bytes of the data for each columns in the row.
 * @return the size in bytes each row needs.
 */
static inline int32_t compute_fixed_width_layout(std::vector<data_type> const &schema,
                                                 std::vector<size_type> &column_start,
                                                 std::vector<size_type> &column_size) {
  // We guarantee that the start of each column is 64-bit aligned so anything can go
  // there, but to make the code simple we will still do an alignment for it.
  int32_t at_offset = 0;
  for (auto col = schema.begin(); col < schema.end(); col++) {
    size_type s = size_of(*col);
    column_size.emplace_back(s);
    std::size_t allocation_needed = s;
    std::size_t alignment_needed = allocation_needed; // They are the same for fixed width types
    at_offset = util::round_up_unsafe(at_offset, static_cast<int32_t>(alignment_needed));
    column_start.emplace_back(at_offset);
    at_offset += allocation_needed;
  }

  // Now we need to add in space for validity
  // Eventually we can think about nullable vs not nullable, but for now we will just always add
  // it in
  int32_t const validity_bytes_needed =
      util::div_rounding_up_safe<int32_t>(schema.size(), CHAR_BIT);
  // validity comes at the end and is byte aligned so we can pack more in.
  at_offset += validity_bytes_needed;
  // Now we need to pad the end so all rows are 64 bit aligned
  return util::round_up_unsafe(at_offset, JCUDF_ROW_ALIGNMENT);
}

/**
 * @brief column sizes and column start offsets for a table
 */
struct column_info_s {
  size_type size_per_row;
  std::vector<size_type> column_starts;
  std::vector<size_type> column_sizes;
  std::vector<size_type> variable_width_column_starts;

  column_info_s &operator=(column_info_s const &other) = delete;
  column_info_s &operator=(column_info_s &&other) = delete;
};

/**
 * @brief Compute information about a table such as bytes per row and offsets.
 *
 * @tparam iterator iterator of column schema data
 * @param begin starting iterator of column schema
 * @param end ending iterator of column schema
 * @param column_starts column start offsets
 * @param column_sizes size in bytes of each column
 * @return size of the fixed_width data portion of a row.
 */
template <typename iterator>
column_info_s compute_column_information(iterator begin, iterator end) {
  size_type size_per_row = 0;
  std::vector<size_type> column_starts;
  std::vector<size_type> column_sizes;
  std::vector<size_type> variable_width_column_starts;

  column_starts.reserve(std::distance(begin, end) + 1);
  column_sizes.reserve(std::distance(begin, end));

  for (auto col_type = begin; col_type != end; ++col_type) {
    bool const compound_type = is_compound(*col_type);

    // a list or string column will write a single uint64 of data here for offset/length
    auto const col_size = compound_type ? sizeof(uint32_t) + sizeof(uint32_t) : size_of(*col_type);

    // align size for this type - They are the same for fixed width types and 4 bytes for variable
    // width length/offset combos
    size_type const alignment_needed = compound_type ? __alignof(uint32_t) : col_size;
    size_per_row = util::round_up_unsafe(size_per_row, alignment_needed);
    if (compound_type) {
      variable_width_column_starts.push_back(size_per_row);
    }
    column_starts.push_back(size_per_row);
    column_sizes.push_back(col_size);
    size_per_row += col_size;
  }

  // add validity offset to the end of fixed_width offsets
  auto validity_offset = size_per_row;
  column_starts.push_back(validity_offset);

  // validity is byte-aligned in the JCUDF format
  size_per_row +=
      util::div_rounding_up_safe(static_cast<size_type>(std::distance(begin, end)), CHAR_BIT);

  return {size_per_row, std::move(column_starts), std::move(column_sizes),
          std::move(variable_width_column_starts)};
}

/**
 * @brief Build `tile_info` for the validity data to break up the work.
 *
 * @param num_columns number of columns in the table
 * @param num_rows number of rows in the table
 * @param shmem_limit_per_tile size of shared memory available to a single gpu tile
 * @param row_batches batched row information for multiple output locations
 * @return vector of `tile_info` structs for validity data
 */
std::vector<detail::tile_info>
build_validity_tile_infos(size_type const &num_columns, size_type const &num_rows,
                          size_type const &shmem_limit_per_tile,
                          std::vector<row_batch> const &row_batches) {
  auto const desired_rows_and_columns = static_cast<int>(sqrt(shmem_limit_per_tile));
  auto const column_stride = util::round_up_unsafe(
      [&]() {
        if (desired_rows_and_columns > num_columns) {
          // not many columns, build a single tile for table width and ship it off
          return num_columns;
        } else {
          return util::round_down_safe(desired_rows_and_columns, CHAR_BIT);
        }
      }(),
      JCUDF_ROW_ALIGNMENT);

  // we fit as much as we can given the column stride note that an element in the table takes just 1
  // bit, but a row with a single element still takes 8 bytes!
  auto const bytes_per_row = util::round_up_safe(
      util::div_rounding_up_unsafe(column_stride, CHAR_BIT), JCUDF_ROW_ALIGNMENT);
  auto const row_stride =
      std::min(num_rows, util::round_down_safe(shmem_limit_per_tile / bytes_per_row, 64));
  std::vector<detail::tile_info> validity_tile_infos;
  validity_tile_infos.reserve(num_columns / column_stride * num_rows / row_stride);
  for (int col = 0; col < num_columns; col += column_stride) {
    int current_tile_row_batch = 0;
    int rows_left_in_batch = row_batches[current_tile_row_batch].row_count;
    int row = 0;
    while (row < num_rows) {
      if (rows_left_in_batch == 0) {
        current_tile_row_batch++;
        rows_left_in_batch = row_batches[current_tile_row_batch].row_count;
      }
      int const tile_height = std::min(row_stride, rows_left_in_batch);
      validity_tile_infos.emplace_back(
          detail::tile_info{col, row, std::min(col + column_stride - 1, num_columns - 1),
                            row + tile_height - 1, current_tile_row_batch});
      row += tile_height;
      rows_left_in_batch -= tile_height;
    }
  }

  return validity_tile_infos;
}

/**
 * @brief functor that returns the size of a row or 0 is row is greater than the number of rows in
 * the table
 *
 * @tparam RowSize iterator that returns the size of a specific row
 */
template <typename RowSize> struct row_size_functor {
  row_size_functor(size_type row_end, RowSize row_sizes, size_type last_row_end)
      : _row_end(row_end), _row_sizes(row_sizes), _last_row_end(last_row_end) {}

  __device__ inline uint64_t operator()(int i) const {
    return i >= _row_end ? 0 : _row_sizes[i + _last_row_end];
  }

  size_type _row_end;
  RowSize _row_sizes;
  size_type _last_row_end;
};

/**
 * @brief Builds batches of rows that will fit in the size limit of a column.
 *
 * @tparam RowSize iterator that gives the size of a specific row of the table.
 * @param num_rows Total number of rows in the table
 * @param row_sizes iterator that gives the size of a specific row of the table.
 * @param all_fixed_width bool indicating all data in this table is fixed width
 * @param stream stream to operate on for this work
 * @param mr memory resource used to allocate any returned data
 * @returns vector of size_type's that indicate row numbers for batch boundaries and a
 * device_uvector of row offsets
 */
template <typename RowSize>
batch_data build_batches(size_type num_rows, RowSize row_sizes, bool all_fixed_width,
                         rmm::cuda_stream_view stream, rmm::mr::device_memory_resource *mr) {
  auto const total_size = thrust::reduce(rmm::exec_policy(stream), row_sizes, row_sizes + num_rows);
  auto const num_batches = static_cast<int32_t>(
      util::div_rounding_up_safe(total_size, static_cast<uint64_t>(MAX_BATCH_SIZE)));
  auto const num_offsets = num_batches + 1;
  std::vector<row_batch> row_batches;
  std::vector<size_type> batch_row_boundaries;
  device_uvector<size_type> batch_row_offsets(all_fixed_width ? 0 : num_rows, stream);

  // at most max gpu memory / 2GB iterations.
  batch_row_boundaries.reserve(num_offsets);
  batch_row_boundaries.push_back(0);
  size_type last_row_end = 0;
  device_uvector<uint64_t> cumulative_row_sizes(num_rows, stream);
  thrust::inclusive_scan(rmm::exec_policy(stream), row_sizes, row_sizes + num_rows,
                         cumulative_row_sizes.begin());

  // This needs to be split this into 2 gig batches. Care must be taken to avoid a batch larger than
  // 2 gigs. Imagine a table with 900 meg rows. The batches should occur every 2 rows, but if a
  // lower bound is run at 2 gigs, 4 gigs, 6 gigs. the batches will be 2 rows, 2 rows, 3 rows, which
  // will be invalid. The previous batch size must be taken into account when building a new batch.
  // One way is to pull the batch size back to the host and add it to MAX_BATCH_SIZE for the lower
  // bound search. The other method involves keeping everything on device, but subtracting the
  // previous batch from cumulative_row_sizes based on index. This involves no synchronization
  // between GPU and CPU, but involves more work on the GPU. These further need to be broken on a
  // 32-row boundary to match the fixed_width optimized versions.

  while (last_row_end < num_rows) {
    auto offset_row_sizes = thrust::make_transform_iterator(
        cumulative_row_sizes.begin(),
        [last_row_end, cumulative_row_sizes = cumulative_row_sizes.data()] __device__(auto i) {
          return i - cumulative_row_sizes[last_row_end];
        });
    auto search_start = offset_row_sizes + last_row_end;
    auto search_end = offset_row_sizes + num_rows;

    // find the next MAX_BATCH_SIZE boundary
    auto const lb =
        thrust::lower_bound(rmm::exec_policy(stream), search_start, search_end, MAX_BATCH_SIZE);
    size_type const batch_size = lb - search_start;

    size_type const row_end = lb == search_end ?
                                  batch_size + last_row_end :
                                  last_row_end + util::round_down_safe(batch_size, 32);

    // build offset list for each row in this batch
    auto const num_rows_in_batch = row_end - last_row_end;

    // build offset list for each row in this batch
    auto const num_entries = row_end - last_row_end + 1;
    device_uvector<size_type> output_batch_row_offsets(num_entries, stream, mr);

    auto row_size_iter_bounded = cudf::detail::make_counting_transform_iterator(
        0, row_size_functor(row_end, row_sizes, last_row_end));

    thrust::exclusive_scan(rmm::exec_policy(stream), row_size_iter_bounded,
                           row_size_iter_bounded + num_entries, output_batch_row_offsets.begin());

    auto const batch_bytes = output_batch_row_offsets.element(num_rows_in_batch, stream);

    // The output_batch_row_offsets vector is used as the offset column of the returned data. This
    // needs to be individually allocated, but the kernel needs a contiguous array of offsets or
    // more global lookups are necessary.
    if (!all_fixed_width) {
      hipMemcpy(batch_row_offsets.data() + last_row_end, output_batch_row_offsets.data(),
                 num_rows_in_batch * sizeof(size_type), hipMemcpyDeviceToDevice);
    }

    batch_row_boundaries.push_back(row_end);
    row_batches.push_back({batch_bytes, num_rows_in_batch, std::move(output_batch_row_offsets)});

    last_row_end = row_end;
  }

  return {std::move(batch_row_offsets), make_device_uvector_async(batch_row_boundaries, stream),
          std::move(batch_row_boundaries), std::move(row_batches)};
}

/**
 * @brief Computes the number of tiles necessary given a tile height and batch offsets
 *
 * @param batch_row_boundaries row boundaries for each batch
 * @param desired_tile_height height of each tile in the table
 * @param stream stream to use
 * @return number of tiles necessary
 */
int compute_tile_counts(device_span<size_type const> const &batch_row_boundaries,
                        int desired_tile_height, rmm::cuda_stream_view stream) {
  size_type const num_batches = batch_row_boundaries.size() - 1;
  device_uvector<size_type> num_tiles(num_batches, stream);
  auto iter = thrust::make_counting_iterator(0);
  thrust::transform(rmm::exec_policy(stream), iter, iter + num_batches, num_tiles.begin(),
                    [desired_tile_height,
                     batch_row_boundaries =
                         batch_row_boundaries.data()] __device__(auto batch_index) -> size_type {
                      return util::div_rounding_up_unsafe(batch_row_boundaries[batch_index + 1] -
                                                              batch_row_boundaries[batch_index],
                                                          desired_tile_height);
                    });
  return thrust::reduce(rmm::exec_policy(stream), num_tiles.begin(), num_tiles.end());
}

/**
 * @brief Builds the `tile_info` structs for a given table.
 *
 * @param tiles span of tiles to populate
 * @param batch_row_boundaries boundary to row batches
 * @param column_start starting column of the tile
 * @param column_end ending column of the tile
 * @param desired_tile_height height of the tile
 * @param total_number_of_rows total number of rows in the table
 * @param stream stream to use
 * @return number of tiles created
 */
size_type
build_tiles(device_span<tile_info> tiles,
            device_uvector<size_type> const &batch_row_boundaries, // comes from build_batches
            int column_start, int column_end, int desired_tile_height, int total_number_of_rows,
            rmm::cuda_stream_view stream) {
  size_type const num_batches = batch_row_boundaries.size() - 1;
  device_uvector<size_type> num_tiles(num_batches, stream);
  auto iter = thrust::make_counting_iterator(0);
  thrust::transform(rmm::exec_policy(stream), iter, iter + num_batches, num_tiles.begin(),
                    [desired_tile_height,
                     batch_row_boundaries =
                         batch_row_boundaries.data()] __device__(auto batch_index) -> size_type {
                      return util::div_rounding_up_unsafe(batch_row_boundaries[batch_index + 1] -
                                                              batch_row_boundaries[batch_index],
                                                          desired_tile_height);
                    });

  size_type const total_tiles =
      thrust::reduce(rmm::exec_policy(stream), num_tiles.begin(), num_tiles.end());

  device_uvector<size_type> tile_starts(num_batches + 1, stream);
  auto tile_iter = cudf::detail::make_counting_transform_iterator(
      0, [num_tiles = num_tiles.data(), num_batches] __device__(auto i) {
        return (i < num_batches) ? num_tiles[i] : 0;
      });
  thrust::exclusive_scan(rmm::exec_policy(stream), tile_iter, tile_iter + num_batches + 1,
                         tile_starts.begin()); // in tiles

  thrust::transform(
      rmm::exec_policy(stream), iter, iter + total_tiles, tiles.begin(),
      [=, tile_starts = tile_starts.data(),
       batch_row_boundaries = batch_row_boundaries.data()] __device__(size_type tile_index) {
        // what batch this tile falls in
        auto const batch_index_iter =
            thrust::upper_bound(thrust::seq, tile_starts, tile_starts + num_batches, tile_index);
        auto const batch_index = std::distance(tile_starts, batch_index_iter) - 1;
        // local index within the tile
        int const local_tile_index = tile_index - tile_starts[batch_index];
        // the start row for this batch.
        int const batch_row_start = batch_row_boundaries[batch_index];
        // the start row for this tile
        int const tile_row_start = batch_row_start + (local_tile_index * desired_tile_height);
        // the end row for this tile
        int const max_row =
            std::min(total_number_of_rows - 1,
                     batch_index + 1 > num_batches ?
                         std::numeric_limits<size_type>::max() :
                         static_cast<int>(batch_row_boundaries[batch_index + 1]) - 1);
        int const tile_row_end =
            std::min(batch_row_start + ((local_tile_index + 1) * desired_tile_height) - 1, max_row);

        // stuff the tile
        return tile_info{column_start, tile_row_start, column_end, tile_row_end,
                         static_cast<int>(batch_index)};
      });

  return total_tiles;
}

/**
 * @brief Determines what data should be operated on by each tile for the incoming table.
 *
 * @tparam TileCallback Callback that receives the start and end columns of tiles
 * @param column_sizes vector of the size of each column
 * @param column_starts vector of the offset of each column
 * @param first_row_batch_size size of the first row batch to limit max tile size since a tile
 * is unable to span batches
 * @param total_number_of_rows total number of rows in the table
 * @param shmem_limit_per_tile shared memory allowed per tile
 * @param f callback function called when building a tile
 */
template <typename TileCallback>
void determine_tiles(std::vector<size_type> const &column_sizes,
                     std::vector<size_type> const &column_starts,
                     size_type const first_row_batch_size, size_type const total_number_of_rows,
                     size_type const &shmem_limit_per_tile, TileCallback f) {
  // tile infos are organized with the tile going "down" the columns this provides the most
  // coalescing of memory access
  int current_tile_width = 0;
  int current_tile_start_col = 0;

  // the ideal tile height has lots of 8-byte reads and 8-byte writes. The optimal read/write would
  // be memory cache line sized access, but since other tiles will read/write the edges this may not
  // turn out to be overly important. For now, we will attempt to build a square tile as far as byte
  // sizes. x * y = shared_mem_size. Which translates to x^2 = shared_mem_size since we want them
  // equal, so height and width are sqrt(shared_mem_size). The trick is that it's in bytes, not rows
  // or columns.
  auto const square_bias = 32; // bias towards columns for performance reasons
  auto const optimal_square_len = static_cast<size_type>(sqrt(shmem_limit_per_tile));
  auto const desired_tile_height = util::round_up_safe<int>(
      std::min(optimal_square_len / square_bias, total_number_of_rows), cudf::detail::warp_size);
  auto const tile_height = std::clamp(desired_tile_height, 1, first_row_batch_size);

  int row_size = 0;

  // march each column and build the tiles of appropriate sizes
  for (uint col = 0; col < column_sizes.size(); ++col) {
    auto const col_size = column_sizes[col];

    // align size for this type
    auto const alignment_needed = col_size; // They are the same for fixed width types
    auto const row_size_aligned = util::round_up_unsafe(row_size, alignment_needed);
    auto const row_size_with_this_col = row_size_aligned + col_size;
    auto const row_size_with_end_pad =
        util::round_up_unsafe(row_size_with_this_col, JCUDF_ROW_ALIGNMENT);

    if (row_size_with_end_pad * tile_height > shmem_limit_per_tile) {
      // too large, close this tile, generate vertical tiles and restart
      f(current_tile_start_col, col == 0 ? col : col - 1, tile_height);

      row_size =
          util::round_up_unsafe((column_starts[col] + column_sizes[col]) & 7, alignment_needed);
      row_size += col_size; // alignment required for shared memory tile boundary to match alignment
                            // of output row
      current_tile_start_col = col;
      current_tile_width = 0;
    } else {
      row_size = row_size_with_this_col;
      current_tile_width++;
    }
  }

  // build last set of tiles
  if (current_tile_width > 0) {
    f(current_tile_start_col, static_cast<int>(column_sizes.size()) - 1, tile_height);
  }
}

/**
 * @brief convert cudf table into JCUDF row format
 *
 * @tparam offsetFunctor functor type for offset functor
 * @param tbl table to convert to JCUDF row format
 * @param batch_info information about the batches of data
 * @param offset_functor functor that returns the starting offset of each row
 * @param column_info information about incoming columns
 * @param variable_width_offsets optional vector of offsets for variable-with columns
 * @param stream stream used
 * @param mr selected memory resource for returned data
 * @return vector of list columns containing byte columns of the JCUDF row data
 */
template <typename offsetFunctor>
std::vector<std::unique_ptr<column>> convert_to_rows(
    table_view const &tbl, batch_data &batch_info, offsetFunctor offset_functor,
    column_info_s const &column_info,
    std::optional<rmm::device_uvector<strings_column_view::offset_iterator>> variable_width_offsets,
    rmm::cuda_stream_view stream, rmm::mr::device_memory_resource *mr) {
  int device_id;
  CUDF_CUDA_TRY(hipGetDevice(&device_id));
  int total_shmem_in_bytes;
  CUDF_CUDA_TRY(
      hipDeviceGetAttribute(&total_shmem_in_bytes, hipDeviceAttributeMaxSharedMemoryPerBlock, device_id));

#ifndef __CUDA_ARCH__ // __host__ code.
  // Need to reduce total shmem available by the size of barriers in the kernel's shared memory
  total_shmem_in_bytes -=
      util::round_up_unsafe(sizeof(cuda::barrier<cuda::thread_scope_block>), 16ul);
#endif // __CUDA_ARCH__

  auto const shmem_limit_per_tile = total_shmem_in_bytes;

  auto const num_rows = tbl.num_rows();
  auto const fixed_width_only = !variable_width_offsets.has_value();

  auto select_columns = [](auto const &tbl, auto column_predicate) {
    std::vector<column_view> cols;
    std::copy_if(tbl.begin(), tbl.end(), std::back_inserter(cols),
                 [&](auto c) { return column_predicate(c); });
    return table_view(cols);
  };

  auto dev_col_sizes = make_device_uvector_async(column_info.column_sizes, stream);
  auto dev_col_starts = make_device_uvector_async(column_info.column_starts, stream);

  // Get the pointers to the input columnar data ready
  auto const data_begin = thrust::make_transform_iterator(tbl.begin(), [](auto const &c) {
    return is_compound(c.type()) ? nullptr : c.template data<int8_t>();
  });
  std::vector<int8_t const *> input_data(data_begin, data_begin + tbl.num_columns());

  // validity code handles variable and fixed-width data, so give it everything
  auto const nm_begin =
      thrust::make_transform_iterator(tbl.begin(), [](auto const &c) { return c.null_mask(); });
  std::vector<bitmask_type const *> input_nm(nm_begin, nm_begin + tbl.num_columns());

  auto dev_input_data = make_device_uvector_async(input_data, stream);
  auto dev_input_nm = make_device_uvector_async(input_nm, stream);

  // the first batch always exists unless we were sent an empty table
  auto const first_batch_size = batch_info.row_batches[0].row_count;

  std::vector<rmm::device_buffer> output_buffers;
  std::vector<int8_t *> output_data;
  output_data.reserve(batch_info.row_batches.size());
  output_buffers.reserve(batch_info.row_batches.size());
  std::transform(batch_info.row_batches.begin(), batch_info.row_batches.end(),
                 std::back_inserter(output_buffers), [&](auto const &batch) {
                   return rmm::device_buffer(batch.num_bytes, stream, mr);
                 });
  std::transform(output_buffers.begin(), output_buffers.end(), std::back_inserter(output_data),
                 [](auto &buf) { return static_cast<int8_t *>(buf.data()); });

  auto dev_output_data = make_device_uvector_async(output_data, stream, mr);

  int info_count = 0;
  detail::determine_tiles(
      column_info.column_sizes, column_info.column_starts, first_batch_size, num_rows,
      shmem_limit_per_tile,
      [&gpu_batch_row_boundaries = batch_info.d_batch_row_boundaries, &info_count,
       &stream](int const start_col, int const end_col, int const tile_height) {
        int i = detail::compute_tile_counts(gpu_batch_row_boundaries, tile_height, stream);
        info_count += i;
      });

  // allocate space for tiles
  device_uvector<detail::tile_info> gpu_tile_infos(info_count, stream);
  int tile_offset = 0;

  detail::determine_tiles(
      column_info.column_sizes, column_info.column_starts, first_batch_size, num_rows,
      shmem_limit_per_tile,
      [&gpu_batch_row_boundaries = batch_info.d_batch_row_boundaries, &gpu_tile_infos, num_rows,
       &tile_offset, stream](int const start_col, int const end_col, int const tile_height) {
        tile_offset += detail::build_tiles(
            {gpu_tile_infos.data() + tile_offset, gpu_tile_infos.size() - tile_offset},
            gpu_batch_row_boundaries, start_col, end_col, tile_height, num_rows, stream);
      });

  // build validity tiles for ALL columns, variable and fixed width.
  auto validity_tile_infos = detail::build_validity_tile_infos(
      tbl.num_columns(), num_rows, shmem_limit_per_tile, batch_info.row_batches);

  auto dev_validity_tile_infos = make_device_uvector_async(validity_tile_infos, stream);

  auto const validity_offset = column_info.column_starts.back();

  // blast through the entire table and convert it
  detail::copy_to_rows<<<gpu_tile_infos.size(), NUM_WARPS_IN_BLOCK * cudf::detail::warp_size,
                         total_shmem_in_bytes, stream.value()>>>(
      num_rows, tbl.num_columns(), shmem_limit_per_tile, gpu_tile_infos, dev_input_data.data(),
      dev_col_sizes.data(), dev_col_starts.data(), offset_functor,
      batch_info.d_batch_row_boundaries.data(),
      reinterpret_cast<int8_t **>(dev_output_data.data()));

  // note that validity gets the entire table and not the fixed-width portion
  detail::copy_validity_to_rows<<<validity_tile_infos.size(),
                                  NUM_WARPS_IN_BLOCK * cudf::detail::warp_size,
                                  total_shmem_in_bytes, stream.value()>>>(
      num_rows, tbl.num_columns(), shmem_limit_per_tile, offset_functor,
      batch_info.d_batch_row_boundaries.data(), dev_output_data.data(), validity_offset,
      dev_validity_tile_infos, dev_input_nm.data());

  if (!fixed_width_only) {
    // build table view for variable-width data only
    auto const variable_width_table =
        select_columns(tbl, [](auto col) { return is_compound(col.type()); });

    CUDF_EXPECTS(!variable_width_table.is_empty(), "No variable-width columns when expected!");
    CUDF_EXPECTS(variable_width_offsets.has_value(), "No variable width offset data!");

    auto const variable_data_begin =
        thrust::make_transform_iterator(variable_width_table.begin(), [](auto const &c) {
          strings_column_view const scv{c};
          return is_compound(c.type()) ? scv.chars().template data<int8_t>() : nullptr;
        });
    std::vector<int8_t const *> variable_width_input_data(
        variable_data_begin, variable_data_begin + variable_width_table.num_columns());

    auto dev_variable_input_data = make_device_uvector_async(variable_width_input_data, stream);
    auto dev_variable_col_output_offsets =
        make_device_uvector_async(column_info.variable_width_column_starts, stream);

    for (uint i = 0; i < batch_info.row_batches.size(); i++) {
      auto const batch_row_offset = batch_info.batch_row_boundaries[i];
      auto const batch_num_rows = batch_info.row_batches[i].row_count;

      dim3 const string_blocks(std::min(
          MAX_STRING_BLOCKS,
          util::div_rounding_up_unsafe(batch_num_rows, NUM_STRING_ROWS_PER_BLOCK_TO_ROWS)));

      detail::copy_strings_to_rows<<<string_blocks, NUM_WARPS_IN_BLOCK * cudf::detail::warp_size, 0,
                                     stream.value()>>>(
          batch_num_rows, variable_width_table.num_columns(), dev_variable_input_data.data(),
          dev_variable_col_output_offsets.data(), variable_width_offsets->data(),
          column_info.size_per_row, offset_functor, batch_row_offset,
          reinterpret_cast<int8_t *>(output_data[i]));
    }
  }

  // split up the output buffer into multiple buffers based on row batch sizes and create list of
  // byte columns
  std::vector<std::unique_ptr<column>> ret;
  ret.reserve(batch_info.row_batches.size());
  auto counting_iter = thrust::make_counting_iterator(0);
  std::transform(counting_iter, counting_iter + batch_info.row_batches.size(),
                 std::back_inserter(ret), [&](auto batch) {
                   auto const offset_count = batch_info.row_batches[batch].row_offsets.size();
                   auto offsets = std::make_unique<column>(
                       data_type{type_id::INT32}, (size_type)offset_count,
                       batch_info.row_batches[batch].row_offsets.release());
                   auto data = std::make_unique<column>(data_type{type_id::INT8},
                                                        batch_info.row_batches[batch].num_bytes,
                                                        std::move(output_buffers[batch]));

                   return make_lists_column(
                       batch_info.row_batches[batch].row_count, std::move(offsets), std::move(data),
                       0, rmm::device_buffer{0, cudf::get_default_stream(), mr}, stream, mr);
                 });

  return ret;
}

} // namespace detail

/**
 * @brief convert a cudf table to JCUDF row format
 *
 * @param tbl incoming table to convert
 * @param stream stream to use for operations
 * @param mr memory resource used for returned data
 * @return vector of list columns containing byte columns of the JCUDF row data
 */
std::vector<std::unique_ptr<column>> convert_to_rows(table_view const &tbl,
                                                     rmm::cuda_stream_view stream,
                                                     rmm::mr::device_memory_resource *mr) {
  auto const num_columns = tbl.num_columns();
  auto const num_rows = tbl.num_rows();

  auto const fixed_width_only = std::all_of(
      tbl.begin(), tbl.end(), [](column_view const &c) { return is_fixed_width(c.type()); });

  // Break up the work into tiles, which are a starting and ending row/col #. This tile size is
  // calculated based on the shared memory size available we want a single tile to fill up the
  // entire shared memory space available for the transpose-like conversion.

  // There are two different processes going on here. The GPU conversion of the data and the writing
  // of the data into the list of byte columns that are a maximum of 2 gigs each due to offset
  // maximum size. The GPU conversion portion has to understand this limitation because the column
  // must own the data inside and as a result it must be a distinct allocation for that column.
  // Copying the data into these final buffers would be prohibitively expensive, so care is taken to
  // ensure the GPU writes to the proper buffer. The tiles are broken at the boundaries of specific
  // rows based on the row sizes up to that point. These are row batches and they are decided first
  // before building the tiles so the tiles can be properly cut around them.

  auto schema_column_iter =
      thrust::make_transform_iterator(tbl.begin(), [](auto const &i) { return i.type(); });

  auto column_info =
      detail::compute_column_information(schema_column_iter, schema_column_iter + num_columns);
  auto const size_per_row = column_info.size_per_row;
  if (fixed_width_only) {
    // total encoded row size. This includes fixed-width data and validity only. It does not include
    // variable-width data since it isn't copied with the fixed-width and validity kernel.
    auto row_size_iter = thrust::make_constant_iterator<uint64_t>(
        util::round_up_unsafe(size_per_row, JCUDF_ROW_ALIGNMENT));

    auto batch_info = detail::build_batches(num_rows, row_size_iter, fixed_width_only, stream, mr);

    detail::fixed_width_row_offset_functor offset_functor(
        util::round_up_unsafe(size_per_row, JCUDF_ROW_ALIGNMENT));

    return detail::convert_to_rows(tbl, batch_info, offset_functor, std::move(column_info),
                                   std::nullopt, stream, mr);
  } else {
    auto offset_data = detail::build_string_row_offsets(tbl, size_per_row, stream);
    auto &row_sizes = std::get<0>(offset_data);

    auto row_size_iter = cudf::detail::make_counting_transform_iterator(
        0, detail::row_size_functor(num_rows, row_sizes.data(), 0));

    auto batch_info = detail::build_batches(num_rows, row_size_iter, fixed_width_only, stream, mr);

    detail::string_row_offset_functor offset_functor(batch_info.batch_row_offsets);

    return detail::convert_to_rows(tbl, batch_info, offset_functor, std::move(column_info),
                                   std::make_optional(std::move(std::get<1>(offset_data))), stream,
                                   mr);
  }
}

std::vector<std::unique_ptr<column>>
convert_to_rows_fixed_width_optimized(table_view const &tbl, rmm::cuda_stream_view stream,
                                      rmm::mr::device_memory_resource *mr) {
  auto const num_columns = tbl.num_columns();

  std::vector<data_type> schema;
  schema.resize(num_columns);
  std::transform(tbl.begin(), tbl.end(), schema.begin(),
                 [](auto i) -> data_type { return i.type(); });

  if (detail::are_all_fixed_width(schema)) {
    std::vector<size_type> column_start;
    std::vector<size_type> column_size;

    int32_t const size_per_row =
        detail::compute_fixed_width_layout(schema, column_start, column_size);
    auto dev_column_start = make_device_uvector_async(column_start, stream, mr);
    auto dev_column_size = make_device_uvector_async(column_size, stream, mr);

    // Make the number of rows per batch a multiple of 32 so we don't have to worry about splitting
    // validity at a specific row offset.  This might change in the future.
    auto const max_rows_per_batch =
        util::round_down_safe(std::numeric_limits<size_type>::max() / size_per_row, 32);

    auto const num_rows = tbl.num_rows();

    // Get the pointers to the input columnar data ready
    std::vector<const int8_t *> input_data;
    std::vector<bitmask_type const *> input_nm;
    for (size_type column_number = 0; column_number < num_columns; column_number++) {
      column_view cv = tbl.column(column_number);
      input_data.emplace_back(cv.data<int8_t>());
      input_nm.emplace_back(cv.null_mask());
    }
    auto dev_input_data = make_device_uvector_async(input_data, stream, mr);
    auto dev_input_nm = make_device_uvector_async(input_nm, stream, mr);

    using ScalarType = scalar_type_t<size_type>;
    auto zero = make_numeric_scalar(data_type(type_id::INT32), stream.value());
    zero->set_valid_async(true, stream);
    static_cast<ScalarType *>(zero.get())->set_value(0, stream);

    auto step = make_numeric_scalar(data_type(type_id::INT32), stream.value());
    step->set_valid_async(true, stream);
    static_cast<ScalarType *>(step.get())->set_value(static_cast<size_type>(size_per_row), stream);

    std::vector<std::unique_ptr<column>> ret;
    for (size_type row_start = 0; row_start < num_rows; row_start += max_rows_per_batch) {
      size_type row_count = num_rows - row_start;
      row_count = row_count > max_rows_per_batch ? max_rows_per_batch : row_count;
      ret.emplace_back(detail::fixed_width_convert_to_rows(
          row_start, row_count, num_columns, size_per_row, dev_column_start, dev_column_size,
          dev_input_data, dev_input_nm, *zero, *step, stream, mr));
    }

    return ret;
  } else {
    CUDF_FAIL("Only fixed width types are currently supported");
  }
}

/**
 * @brief convert from JCUDF row format to cudf columns
 *
 * @param input vector of list columns containing byte columns of the JCUDF row data
 * @param schema incoming schema of the data
 * @param stream stream to use for compute
 * @param mr memory resource for returned data
 * @return cudf table of the data
 */
std::unique_ptr<table> convert_from_rows(lists_column_view const &input,
                                         std::vector<data_type> const &schema,
                                         rmm::cuda_stream_view stream,
                                         rmm::mr::device_memory_resource *mr) {
  // verify that the types are what we expect
  column_view child = input.child();
  auto const list_type = child.type().id();
  CUDF_EXPECTS(list_type == type_id::INT8 || list_type == type_id::UINT8,
               "Only a list of bytes is supported as input");

  // convert any strings in the schema to two int32 columns
  // This allows us to leverage the fixed-width copy code to fill in our offset and string length
  // data.
  std::vector<data_type> string_schema;
  string_schema.reserve(schema.size());
  for (auto i : schema) {
    if (i.id() == type_id::STRING) {
      string_schema.push_back(data_type(type_id::INT32));
      string_schema.push_back(data_type(type_id::INT32));
    } else {
      string_schema.push_back(i);
    }
  }

  auto const num_columns = string_schema.size();
  auto const num_rows = input.parent().size();

  int device_id;
  CUDF_CUDA_TRY(hipGetDevice(&device_id));
  int total_shmem_in_bytes;
  CUDF_CUDA_TRY(
      hipDeviceGetAttribute(&total_shmem_in_bytes, hipDeviceAttributeMaxSharedMemoryPerBlock, device_id));

#ifndef __CUDA_ARCH__ // __host__ code.
  // Need to reduce total shmem available by the size of barriers in the kernel's shared memory
  total_shmem_in_bytes -=
      util::round_up_unsafe(sizeof(cuda::barrier<cuda::thread_scope_block>), 16ul);
#endif // __CUDA_ARCH__

  auto const shmem_limit_per_tile = total_shmem_in_bytes;

  auto column_info = detail::compute_column_information(string_schema.begin(), string_schema.end());
  auto const size_per_row = util::round_up_unsafe(column_info.size_per_row, JCUDF_ROW_ALIGNMENT);

  // Ideally we would check that the offsets are all the same, etc. but for now this is probably
  // fine
  CUDF_EXPECTS(size_per_row * num_rows <= child.size(), "The layout of the data appears to be off");
  auto dev_col_starts = make_device_uvector_async(column_info.column_starts, stream);
  auto dev_col_sizes = make_device_uvector_async(column_info.column_sizes, stream);

  // Allocate the columns we are going to write into
  std::vector<std::unique_ptr<column>> output_columns;
  std::vector<std::unique_ptr<column>> string_row_offset_columns;
  std::vector<std::unique_ptr<column>> string_length_columns;
  std::vector<int8_t *> output_data;
  std::vector<bitmask_type *> output_nm;
  std::vector<int32_t *> string_row_offsets;
  std::vector<int32_t *> string_lengths;
  for (auto i : schema) {
    auto make_col = [&output_data, &output_nm](data_type type, size_type num_rows, bool include_nm,
                                               rmm::cuda_stream_view stream,
                                               rmm::mr::device_memory_resource *mr) {
      auto column = make_fixed_width_column(
          type, num_rows, include_nm ? mask_state::UNINITIALIZED : mask_state::UNALLOCATED, stream,
          mr);
      auto mut = column->mutable_view();
      output_data.emplace_back(mut.data<int8_t>());
      if (include_nm) {
        output_nm.emplace_back(mut.null_mask());
      }
      return column;
    };
    if (i.id() == type_id::STRING) {
      auto const int32type = data_type(type_id::INT32);
      auto offset_col =
          make_col(int32type, num_rows, true, stream, rmm::mr::get_current_device_resource());
      string_row_offsets.push_back(offset_col->mutable_view().data<int32_t>());
      string_row_offset_columns.emplace_back(std::move(offset_col));
      auto length_col =
          make_col(int32type, num_rows, false, stream, rmm::mr::get_current_device_resource());
      string_lengths.push_back(length_col->mutable_view().data<int32_t>());
      string_length_columns.emplace_back(std::move(length_col));
      // placeholder
      output_columns.emplace_back(make_empty_column(type_id::STRING));
    } else {
      output_columns.emplace_back(make_col(i, num_rows, true, stream, mr));
    }
  }

  auto dev_string_row_offsets = make_device_uvector_async(string_row_offsets, stream);
  auto dev_string_lengths = make_device_uvector_async(string_lengths, stream);

  // build the row_batches from the passed in list column
  std::vector<detail::row_batch> row_batches;
  row_batches.push_back(
      {detail::row_batch{child.size(), num_rows, device_uvector<size_type>(0, stream)}});

  auto dev_output_data = make_device_uvector_async(output_data, stream);
  auto dev_output_nm = make_device_uvector_async(output_nm, stream);

  // only ever get a single batch when going from rows, so boundaries are 0, num_rows
  constexpr auto num_batches = 2;
  device_uvector<size_type> gpu_batch_row_boundaries(num_batches, stream);

  thrust::transform(rmm::exec_policy(stream), thrust::make_counting_iterator(0),
                    thrust::make_counting_iterator(num_batches), gpu_batch_row_boundaries.begin(),
                    [num_rows] __device__(auto i) { return i == 0 ? 0 : num_rows; });

  int info_count = 0;
  detail::determine_tiles(
      column_info.column_sizes, column_info.column_starts, num_rows, num_rows, shmem_limit_per_tile,
      [&gpu_batch_row_boundaries, &info_count, &stream](int const start_col, int const end_col,
                                                        int const tile_height) {
        info_count += detail::compute_tile_counts(gpu_batch_row_boundaries, tile_height, stream);
      });

  // allocate space for tiles
  device_uvector<detail::tile_info> gpu_tile_infos(info_count, stream);

  int tile_offset = 0;
  detail::determine_tiles(
      column_info.column_sizes, column_info.column_starts, num_rows, num_rows, shmem_limit_per_tile,
      [&gpu_batch_row_boundaries, &gpu_tile_infos, num_rows, &tile_offset,
       stream](int const start_col, int const end_col, int const tile_height) {
        tile_offset += detail::build_tiles(
            {gpu_tile_infos.data() + tile_offset, gpu_tile_infos.size() - tile_offset},
            gpu_batch_row_boundaries, start_col, end_col, tile_height, num_rows, stream);
      });

  dim3 const blocks(gpu_tile_infos.size());

  // validity needs to be calculated based on the actual number of final table columns
  auto validity_tile_infos =
      detail::build_validity_tile_infos(schema.size(), num_rows, shmem_limit_per_tile, row_batches);

  auto dev_validity_tile_infos = make_device_uvector_async(validity_tile_infos, stream);

  dim3 const validity_blocks(validity_tile_infos.size());

  if (dev_string_row_offsets.size() == 0) {
    detail::fixed_width_row_offset_functor offset_functor(size_per_row);

    detail::copy_from_rows<<<gpu_tile_infos.size(), NUM_WARPS_IN_BLOCK * cudf::detail::warp_size,
                             total_shmem_in_bytes, stream.value()>>>(
        num_rows, num_columns, shmem_limit_per_tile, offset_functor,
        gpu_batch_row_boundaries.data(), dev_output_data.data(), dev_col_sizes.data(),
        dev_col_starts.data(), gpu_tile_infos, child.data<int8_t>());

    detail::copy_validity_from_rows<<<validity_tile_infos.size(),
                                      NUM_WARPS_IN_BLOCK * cudf::detail::warp_size,
                                      total_shmem_in_bytes, stream.value()>>>(
        num_rows, num_columns, shmem_limit_per_tile, offset_functor,
        gpu_batch_row_boundaries.data(), dev_output_nm.data(), column_info.column_starts.back(),
        dev_validity_tile_infos, child.data<int8_t>());

  } else {
    detail::string_row_offset_functor offset_functor(device_span<size_type const>{input.offsets()});
    detail::copy_from_rows<<<gpu_tile_infos.size(), NUM_WARPS_IN_BLOCK * cudf::detail::warp_size,
                             total_shmem_in_bytes, stream.value()>>>(
        num_rows, num_columns, shmem_limit_per_tile, offset_functor,
        gpu_batch_row_boundaries.data(), dev_output_data.data(), dev_col_sizes.data(),
        dev_col_starts.data(), gpu_tile_infos, child.data<int8_t>());

    detail::copy_validity_from_rows<<<validity_tile_infos.size(),
                                      NUM_WARPS_IN_BLOCK * cudf::detail::warp_size,
                                      total_shmem_in_bytes, stream.value()>>>(
        num_rows, num_columns, shmem_limit_per_tile, offset_functor,
        gpu_batch_row_boundaries.data(), dev_output_nm.data(), column_info.column_starts.back(),
        dev_validity_tile_infos, child.data<int8_t>());

    std::vector<device_uvector<size_type>> string_col_offsets;
    std::vector<rmm::device_uvector<char>> string_data_cols;
    std::vector<size_type *> string_col_offset_ptrs;
    std::vector<char *> string_data_col_ptrs;
    for (auto &col_string_lengths : string_lengths) {
      device_uvector<size_type> output_string_offsets(num_rows + 1, stream, mr);
      auto tmp = [num_rows, col_string_lengths] __device__(auto const &i) {
        return i < num_rows ? col_string_lengths[i] : 0;
      };
      auto bounded_iter = cudf::detail::make_counting_transform_iterator(0, tmp);
      thrust::exclusive_scan(rmm::exec_policy(stream), bounded_iter, bounded_iter + num_rows + 1,
                             output_string_offsets.begin());

      // allocate destination string column
      rmm::device_uvector<char> string_data(output_string_offsets.element(num_rows, stream), stream,
                                            mr);

      string_col_offset_ptrs.push_back(output_string_offsets.data());
      string_data_col_ptrs.push_back(string_data.data());
      string_col_offsets.push_back(std::move(output_string_offsets));
      string_data_cols.push_back(std::move(string_data));
    }
    auto dev_string_col_offsets = make_device_uvector_async(string_col_offset_ptrs, stream);
    auto dev_string_data_cols = make_device_uvector_async(string_data_col_ptrs, stream);

    dim3 const string_blocks(
        std::min(std::max(MIN_STRING_BLOCKS, num_rows / NUM_STRING_ROWS_PER_BLOCK_FROM_ROWS),
                 MAX_STRING_BLOCKS));

    detail::copy_strings_from_rows<<<string_blocks, NUM_WARPS_IN_BLOCK * cudf::detail::warp_size, 0,
                                     stream.value()>>>(
        offset_functor, dev_string_row_offsets.data(), dev_string_lengths.data(),
        dev_string_col_offsets.data(), dev_string_data_cols.data(), child.data<int8_t>(), num_rows,
        static_cast<cudf::size_type>(string_col_offsets.size()));

    // merge strings back into output_columns
    int string_idx = 0;
    for (int i = 0; i < static_cast<int>(schema.size()); ++i) {
      if (schema[i].id() == type_id::STRING) {
        // stuff real string column
        auto string_data = string_row_offset_columns[string_idx].release()->release();
        output_columns[i] = make_strings_column(num_rows, std::move(string_col_offsets[string_idx]),
                                                std::move(string_data_cols[string_idx]),
                                                std::move(*string_data.null_mask.release()),
                                                cudf::UNKNOWN_NULL_COUNT);
        string_idx++;
      }
    }
  }

  return std::make_unique<table>(std::move(output_columns));
}

std::unique_ptr<table> convert_from_rows_fixed_width_optimized(
    lists_column_view const &input, std::vector<data_type> const &schema,
    rmm::cuda_stream_view stream, rmm::mr::device_memory_resource *mr) {
  // verify that the types are what we expect
  column_view child = input.child();
  auto const list_type = child.type().id();
  CUDF_EXPECTS(list_type == type_id::INT8 || list_type == type_id::UINT8,
               "Only a list of bytes is supported as input");

  auto const num_columns = schema.size();

  if (detail::are_all_fixed_width(schema)) {
    std::vector<size_type> column_start;
    std::vector<size_type> column_size;

    auto const num_rows = input.parent().size();
    auto const size_per_row = detail::compute_fixed_width_layout(schema, column_start, column_size);

    // Ideally we would check that the offsets are all the same, etc. but for now this is probably
    // fine
    CUDF_EXPECTS(size_per_row * num_rows == child.size(),
                 "The layout of the data appears to be off");
    auto dev_column_start = make_device_uvector_async(column_start, stream);
    auto dev_column_size = make_device_uvector_async(column_size, stream);

    // Allocate the columns we are going to write into
    std::vector<std::unique_ptr<column>> output_columns;
    std::vector<int8_t *> output_data;
    std::vector<bitmask_type *> output_nm;
    for (int i = 0; i < static_cast<int>(num_columns); i++) {
      auto column =
          make_fixed_width_column(schema[i], num_rows, mask_state::UNINITIALIZED, stream, mr);
      auto mut = column->mutable_view();
      output_data.emplace_back(mut.data<int8_t>());
      output_nm.emplace_back(mut.null_mask());
      output_columns.emplace_back(std::move(column));
    }

    auto dev_output_data = make_device_uvector_async(output_data, stream, mr);
    auto dev_output_nm = make_device_uvector_async(output_nm, stream, mr);

    dim3 blocks;
    dim3 threads;
    int shared_size =
        detail::calc_fixed_width_kernel_dims(num_columns, num_rows, size_per_row, blocks, threads);

    detail::copy_from_rows_fixed_width_optimized<<<blocks, threads, shared_size, stream.value()>>>(
        num_rows, num_columns, size_per_row, dev_column_start.data(), dev_column_size.data(),
        dev_output_data.data(), dev_output_nm.data(), child.data<int8_t>());

    return std::make_unique<table>(std::move(output_columns));
  } else {
    CUDF_FAIL("Only fixed width types are currently supported");
  }
}

} // namespace jni

} // namespace cudf
