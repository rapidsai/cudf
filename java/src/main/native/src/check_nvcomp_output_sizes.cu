/*
 * Copyright (c) 2021, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */
#include <cudf/detail/nvtx/nvtx3.hpp>
#include <cudf/utilities/error.hpp>
#include <thrust/device_ptr.h>
#include <thrust/equal.h>

#include "check_nvcomp_output_sizes.hpp"

namespace {

struct java_domain {
  static constexpr char const *name{"Java"};
};

} // anonymous namespace

namespace cudf {
namespace java {

/**
 * Check that the vector of expected uncompressed sizes matches the vector of actual compressed
 * sizes. Both vectors are assumed to be in device memory and contain num_chunks elements.
 */
bool check_nvcomp_output_sizes(std::size_t const *dev_uncompressed_sizes,
                               std::size_t const *dev_actual_uncompressed_sizes,
                               std::size_t num_chunks, rmm::cuda_stream_view stream) {
  NVTX3_FUNC_RANGE_IN(java_domain);
  return thrust::equal(rmm::exec_policy(stream), dev_uncompressed_sizes,
                       dev_uncompressed_sizes + num_chunks, dev_actual_uncompressed_sizes);
}

} // namespace java
} // namespace cudf
