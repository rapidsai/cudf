#include <gdf/gdf.h>
#include <gdf/utils.h>
#include <gdf/errorutils.h>

#include "rmm.h"

#include <cub/device/device_scan.cuh>



template <class T>
struct Scan {
    static
    gdf_error call(const T *inp, T *out, size_t size, bool inclusive) {
        using hipcub::DeviceScan;

        auto scan_function = (inclusive? inclusive_sum : exclusive_sum);

        // Prepare temp storage
        void *temp_storage = NULL;
        size_t temp_storage_bytes = 0;
        scan_function(temp_storage, temp_storage_bytes, inp, out, size);
        RMM_TRY( rmmAlloc(&temp_storage, temp_storage_bytes, 0) ); // TODO: non-default stream
        // Do scan
        scan_function(temp_storage, temp_storage_bytes, inp, out, size);
        // Cleanup
        RMM_TRY( rmmFree(temp_storage, 0) ); // TODO: non-default stream

        return GDF_SUCCESS;
    }

    static
    gdf_error exclusive_sum(void *&temp_storage, size_t &temp_storage_bytes,
                            const T *inp, T *out, size_t size) {
        hipcub::DeviceScan::ExclusiveSum(temp_storage, temp_storage_bytes,
                                      inp, out, size);
        CUDA_CHECK_LAST();
        return GDF_SUCCESS;
    }

    static
    gdf_error inclusive_sum(void *&temp_storage, size_t &temp_storage_bytes,
                            const T *inp, T *out, size_t size) {
        hipcub::DeviceScan::InclusiveSum(temp_storage, temp_storage_bytes,
                                      inp, out, size);
        CUDA_CHECK_LAST();
        return GDF_SUCCESS;
    }
};

#define SCAN_IMPL(F, T)                                                       \
gdf_error gdf_prefixsum_##F(gdf_column *inp, gdf_column *out, int inclusive) {\
    GDF_REQUIRE( inp->size == out->size, GDF_COLUMN_SIZE_MISMATCH );          \
    GDF_REQUIRE( inp->dtype == out->dtype, GDF_UNSUPPORTED_DTYPE );           \
    GDF_REQUIRE( !inp->valid || !inp->null_count, GDF_VALIDITY_UNSUPPORTED ); \
    GDF_REQUIRE( !out->valid || !out->null_count, GDF_VALIDITY_UNSUPPORTED ); \
    return Scan<T>::call((const T*)inp->data, (T*)out->data, inp->size,       \
                         inclusive);                                          \
}


SCAN_IMPL(i8,  int8_t)
SCAN_IMPL(i32, int32_t)
SCAN_IMPL(i64, int64_t)


gdf_error gdf_prefixsum_generic(gdf_column *inp, gdf_column *out,
                                int inclusive)
{
    switch (inp->dtype) {
    case GDF_INT8:    return gdf_prefixsum_i8(inp, out, inclusive);
    case GDF_INT32:   return gdf_prefixsum_i32(inp, out, inclusive);
    case GDF_INT64:   return gdf_prefixsum_i64(inp, out, inclusive);
    default: return GDF_SUCCESS;
    }
}
