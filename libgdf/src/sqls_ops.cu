/* Copyright 2018 NVIDIA Corporation.  All rights reserved. */

//Type-erasure C-style interface for Multi-column Filter, Order-By, and Group-By functionality

#include <gdf/gdf.h>
#include <gdf/utils.h>
#include <gdf/errorutils.h>

#include "thrust_rmm_allocator.h"

///#include "../include/sqls_rtti_comp.hpp" -- CORRECT: put me back
#include "sqls_rtti_comp.hpp"
#include "groupby/groupby.cuh"
#include "groupby/hash/aggregation_operations.cuh"
#include "nvtx_utils.h"

//using IndexT = int;//okay...
using IndexT = size_t;

namespace{ //annonymus

  //helper functions:
  //
  //flatten AOS info from gdf_columns into SOA (2 arrays):
  //(1) column array pointers and (2) types;
  //
  void soa_col_info(gdf_column* cols, size_t ncols, void** d_cols, int* d_types)
  {
    std::vector<void*> v_cols(ncols,nullptr);
    std::vector<int>   v_types(ncols, 0);
    for(size_t i=0;i<ncols;++i)
      {
        v_cols[i] = cols[i].data;
        v_types[i] = cols[i].dtype;
      }

    void** h_cols = &v_cols[0];
    int* h_types = &v_types[0];
    hipMemcpy(d_cols, h_cols, ncols*sizeof(void*), hipMemcpyHostToDevice);//TODO: add streams
    hipMemcpy(d_types, h_types, ncols*sizeof(int), hipMemcpyHostToDevice);//TODO: add streams
  }

  // thrust::device_vector set to use rmmAlloc and rmmFree.
  template<typename T>
  using Vector = thrust::device_vector<T, rmm_allocator<T>>;

  void type_dispatcher(gdf_dtype col_type,
                       int col_index,
                       gdf_column** h_cols_in,
                       gdf_column** h_cols_out,
                       IndexT* d_indices,
                       size_t nrows_new)
  {
    hipStream_t stream = 0; // TODO: non-default stream
    rmm_temp_allocator allocator(stream); 
    auto exec = thrust::cuda::par(allocator).on(stream);

    switch( col_type )
      {
      case GDF_INT8:
        {
          using ColType = int8_t;

          ColType* d_in  = static_cast<ColType*>(h_cols_in[col_index]->data);//pointer semantics (2)
          ColType* d_out = static_cast<ColType*>(h_cols_out[col_index]->data);
          thrust::gather(exec,
                         d_indices, d_indices + nrows_new, //map of indices
                         d_in,                             //source
                         d_out);                           //=source[map]
          break;
        }
      case GDF_INT16:
        {
          using ColType = int16_t;

          ColType* d_in  = static_cast<ColType*>(h_cols_in[col_index]->data);
          ColType* d_out = static_cast<ColType*>(h_cols_out[col_index]->data);
          thrust::gather(exec,
                         d_indices, d_indices + nrows_new, //map of indices
                         d_in,                             //source
                         d_out);                           //=source[map]
          break;
        }
      case GDF_INT32:
        {
          using ColType = int32_t;

          ColType* d_in  = static_cast<ColType*>(h_cols_in[col_index]->data);
          ColType* d_out = static_cast<ColType*>(h_cols_out[col_index]->data);
          thrust::gather(exec,
                         d_indices, d_indices + nrows_new, //map of indices
                         d_in,                             //source
                         d_out);                           //=source[map]
          break;
        }
      case GDF_INT64:
        {
          using ColType = int64_t;

          ColType* d_in  = static_cast<ColType*>(h_cols_in[col_index]->data);
          ColType* d_out = static_cast<ColType*>(h_cols_out[col_index]->data);
          thrust::gather(exec,
                         d_indices, d_indices + nrows_new, //map of indices
                         d_in,                             //source
                         d_out);                           //=source[map]
          break;
        }
      case GDF_FLOAT32:
        {
          using ColType = float;

          ColType* d_in  = static_cast<ColType*>(h_cols_in[col_index]->data);
          ColType* d_out = static_cast<ColType*>(h_cols_out[col_index]->data);
          thrust::gather(exec,
                         d_indices, d_indices + nrows_new, //map of indices
                         d_in,                             //source
                         d_out);                           //=source[map]
          break;
        }
      case GDF_FLOAT64:
        {
          using ColType = double;

          ColType* d_in  = static_cast<ColType*>(h_cols_in[col_index]->data);
          ColType* d_out = static_cast<ColType*>(h_cols_out[col_index]->data);
          thrust::gather(exec,
                         d_indices, d_indices + nrows_new, //map of indices
                         d_in,                             //source
                         d_out);                           //=source[map]
          break;
        }

      default:
        assert( false );//type not handled
      }
    return;// State::True;
  }

  //copy from a set of gdf_columns:    h_cols_in
  //of size (#ncols):                  ncols
  //to another set of columns        : h_cols_out
  //by gathering via array of indices: d_indices
  //of size:                           nrows_new
  //
  void multi_gather_host(size_t ncols,  gdf_column** h_cols_in, gdf_column** h_cols_out, IndexT* d_indices, size_t nrows_new)
  {
    for(size_t col_index = 0; col_index<ncols; ++col_index)
      {
        gdf_dtype col_type = h_cols_in[col_index]->dtype;
        type_dispatcher(col_type,
                        col_index,
                        h_cols_in,
                        h_cols_out,
                        d_indices,
                        nrows_new);

        h_cols_out[col_index]->dtype = col_type;
        h_cols_out[col_index]->size = nrows_new;
        
        //TODO: h_cols_out[col_index]->valid
      }
  }

  int dtype_size(gdf_dtype col_type)
  {
    switch( col_type )
      {
      case GDF_INT8:
        {
          using ColType = int8_t;
	  
          return sizeof(ColType);
        }
      case GDF_INT16:
        {
          using ColType = int16_t;

          return sizeof(ColType);
        }
      case GDF_INT32:
        {
          using ColType = int32_t;

          return sizeof(ColType);
        }
      case GDF_INT64:
        {
          using ColType = int64_t;

          return sizeof(ColType);
        }
      case GDF_FLOAT32:
        {
          using ColType = float;

          return sizeof(ColType);
        }
      case GDF_FLOAT64:
        {
          using ColType = double;

          return sizeof(ColType);
        }

      default:
        assert( false );//type not handled
      }
      return 0;
  }

#ifdef DEBUG_
  void run_echo(size_t nrows,     //in: # rows
                gdf_column* cols, //in: host-side array of gdf_columns
                size_t ncols,     //in: # cols
                int flag_sorted,  //in: flag specifying if rows are pre-sorted (1) or not (0)
                gdf_column agg_in)//in: column to aggregate
  {
    std::cout<<"############# Echo: #############\n";
    std::cout<<"nrows: "<<nrows<<"\n";
    std::cout<<"ncols: "<<ncols<<"\n";
    std::cout<<"sorted: "<<flag_sorted<<"\n";

    std::cout<<"input cols:\n";
    for(auto i = 0; i < ncols; ++i)
      {
        switch(i)
          {
          case 0:
          case 1:
            {
              std::vector<int32_t> v(nrows);
              int32_t* p = &v[0];
              hipMemcpy(p, cols[i].data, nrows*sizeof(int32_t), hipMemcpyDeviceToHost);
              std::copy(v.begin(), v.end(), std::ostream_iterator<int32_t>(std::cout,","));
              std::cout<<"\n";
              break;
            }
          case 2:
            {
              std::vector<double> v(nrows);
              double* p = &v[0];
              hipMemcpy(p, cols[i].data, nrows*sizeof(double), hipMemcpyDeviceToHost);
              std::copy(v.begin(), v.end(), std::ostream_iterator<double>(std::cout,","));
              std::cout<<"\n";
              break;
            }
          }
      }


    std::cout<<"col to aggregate on:\n";
    std::vector<double> v(nrows);
    double* p = &v[0];
    hipMemcpy(p, agg_in.data, nrows*sizeof(double), hipMemcpyDeviceToHost);
    std::copy(v.begin(), v.end(), std::ostream_iterator<double>(std::cout,","));
    std::cout<<"\n";
  }
#endif
  





//apparent duplication of info between
//gdf_column array and two arrays:
//           d_cols = data slice of gdf_column array;
//           d_types = dtype slice of gdf_column array;
//but it's nevessary because the gdf_column array is host
//(even though its data slice is on device)
//
gdf_error gdf_group_by_count(size_t nrows,     //in: # rows
                             gdf_column* cols, //in: host-side array of gdf_columns
                             size_t ncols,     //in: # cols
                             int flag_sorted,  //in: flag specififying if rows are pre-sorted (1) or not (0)
                             void** d_cols,    //out: pre-allocated device-side array to be filled with gdf_column::data for each column; slicing of gdf_column array (host)
                             int* d_types,     //out: pre-allocated device-side array to be filled with gdf_colum::dtype for each column; slicing of gdf_column array (host)
                             IndexT* d_indx,      //out: device-side array of row indices after sorting
                             IndexT* d_kout,      //out: device-side array of rows after gropu-by
                             gdf_column& c_vout,  //out: aggregated column; requires shallow (trivial) copy-construction (see static_assert below);
                             size_t* new_sz,   //out: host-side # rows of d_count
                             bool flag_distinct = false)
{
  //copy H-D:
  //
  soa_col_info(cols, ncols, d_cols, d_types);

  switch( c_vout.dtype )
    {
    case GDF_INT8:
      {
        using T = char;

        T* d_vout  = static_cast<T*>(c_vout.data);
        *new_sz = multi_col_group_by_count_sort(nrows,
                                                ncols,
                                                d_cols,
                                                d_types,
                                                d_indx,
                                                d_kout,
                                                d_vout,
                                                flag_sorted,
                                                flag_distinct);
        
        break;
      }

    case GDF_INT16:
      {
        using T = short;

        T* d_vout  = static_cast<T*>(c_vout.data);
        *new_sz = multi_col_group_by_count_sort(nrows,
                                                ncols,
                                                d_cols,
                                                d_types,
                                                d_indx,
                                                d_kout,
                                                d_vout,
                                                flag_sorted,
                                                flag_distinct);
        
        break;
      }
    case GDF_INT32:
      {
        using T = int;

        T* d_vout  = static_cast<T*>(c_vout.data);
        *new_sz = multi_col_group_by_count_sort(nrows,
                                                ncols,
                                                d_cols,
                                                d_types,
                                                d_indx,
                                                d_kout,
                                                d_vout,
                                                flag_sorted,
                                                flag_distinct);
	
        break;
      }

    case GDF_INT64:
      {
        using T = long;

        T* d_vout  = static_cast<T*>(c_vout.data);
        *new_sz = multi_col_group_by_count_sort(nrows,
                                                ncols,
                                                d_cols,
                                                d_types,
                                                d_indx,
                                                d_kout,
                                                d_vout,
                                                flag_sorted,
                                                flag_distinct);
	
        break;
      }

    case GDF_FLOAT32:
      {
        using T = float;

        T* d_vout  = static_cast<T*>(c_vout.data);
        *new_sz = multi_col_group_by_count_sort(nrows,
                                                ncols,
                                                d_cols,
                                                d_types,
                                                d_indx,
                                                d_kout,
                                                d_vout,
                                                flag_sorted,
                                                flag_distinct);
	
        break;
      }

    case GDF_FLOAT64:
      {
        using T = double;

        T* d_vout  = static_cast<T*>(c_vout.data);
        *new_sz = multi_col_group_by_count_sort(nrows,
                                                ncols,
                                                d_cols,
                                                d_types,
                                                d_indx,
                                                d_kout,
                                                d_vout,
                                                flag_sorted,
                                                flag_distinct);
	
        break;
      }

    default:
      return GDF_UNSUPPORTED_DTYPE;
    }
  
  return GDF_SUCCESS;
}

//apparent duplication of info between
//gdf_column array and two arrays:
//           d_cols = data slice of gdf_column array;
//           d_types = dtype slice of gdf_column array;
//but it's necessary because the gdf_column array is host
//(even though its data slice is on device)
//
gdf_error gdf_group_by_sum(size_t nrows,     //in: # rows
                           gdf_column* cols, //in: host-side array of gdf_columns
                           size_t ncols,     //in: # cols
                           int flag_sorted,  //in: flag specififying if rows are pre-sorted (1) or not (0)
                           gdf_column& agg_in,//in: column to aggregate
                           void** d_cols,    //out: pre-allocated device-side array to be filled with gdf_column::data for each column; slicing of gdf_column array (host)
                           int* d_types,     //out: pre-allocated device-side array to be filled with gdf_colum::dtype for each column; slicing of gdf_column array (host)
                           IndexT* d_indx,      //out: device-side array of row indices after sorting
                           gdf_column& agg_p, //out: reordering of d_agg after sorting; requires shallow (trivial) copy-construction (see static_assert below);
                           IndexT* d_kout,      //out: device-side array of rows after group-by
                           gdf_column& c_vout,//out: aggregated column; requires shallow (trivial) copy-construction (see static_assert below);
                           size_t* new_sz)   //out: host-side # rows of d_count
{
  //not supported by g++-4.8:
  //
  //static_assert(std::is_trivially_copy_constructible<gdf_column>::value,
  //		"error: gdf_column must have shallow copy constructor; otherwise cannot pass output by copy.");

#ifdef DEBUG_
  run_echo(nrows,     //in: # rows
           cols, //in: host-side array of gdf_columns
           ncols,     //in: # cols
           flag_sorted,  //in: flag specififying if rows are pre-sorted (1) or not (0)
           agg_in);//in: column to aggregate
#endif

  assert( agg_in.dtype == agg_p.dtype );
  assert( agg_in.dtype == c_vout.dtype );
  
  //copy H-D:
  //
  soa_col_info(cols, ncols, d_cols, d_types);

  switch( agg_in.dtype )
    {
    case GDF_INT8:
      {
        using T = char;

        T* d_agg   = static_cast<T*>(agg_in.data);
        T* d_agg_p = static_cast<T*>(agg_p.data);
        T* d_vout  = static_cast<T*>(c_vout.data);
        *new_sz = multi_col_group_by_sum_sort(nrows,
                                              ncols,
                                              d_cols,
                                              d_types,
                                              d_agg,
                                              d_indx,
                                              d_agg_p,
                                              d_kout,
                                              d_vout,
                                              flag_sorted);
	
        break;
      }

    case GDF_INT16:
      {
        using T = short;

        T* d_agg   = static_cast<T*>(agg_in.data);
        T* d_agg_p = static_cast<T*>(agg_p.data);
        T* d_vout  = static_cast<T*>(c_vout.data);
        *new_sz = multi_col_group_by_sum_sort(nrows,
                                              ncols,
                                              d_cols,
                                              d_types,
                                              d_agg,
                                              d_indx,
                                              d_agg_p,
                                              d_kout,
                                              d_vout,
                                              flag_sorted);

        break;
      }
    case GDF_INT32:
      {
        using T = int;

        T* d_agg   = static_cast<T*>(agg_in.data);
        T* d_agg_p = static_cast<T*>(agg_p.data);
        T* d_vout  = static_cast<T*>(c_vout.data);
        *new_sz = multi_col_group_by_sum_sort(nrows,
                                              ncols,
                                              d_cols,
                                              d_types,
                                              d_agg,
                                              d_indx,
                                              d_agg_p,
                                              d_kout,
                                              d_vout,
                                              flag_sorted);
	
        break;
      }

    case GDF_INT64:
      {
        using T = long;

        T* d_agg   = static_cast<T*>(agg_in.data);
        T* d_agg_p = static_cast<T*>(agg_p.data);
        T* d_vout  = static_cast<T*>(c_vout.data);
        *new_sz = multi_col_group_by_sum_sort(nrows,
                                              ncols,
                                              d_cols,
                                              d_types,
                                              d_agg,
                                              d_indx,
                                              d_agg_p,
                                              d_kout,
                                              d_vout,
                                              flag_sorted);
	
        break;
      }

    case GDF_FLOAT32:
      {
        using T = float;

        T* d_agg   = static_cast<T*>(agg_in.data);
        T* d_agg_p = static_cast<T*>(agg_p.data);
        T* d_vout  = static_cast<T*>(c_vout.data);
        *new_sz = multi_col_group_by_sum_sort(nrows,
                                              ncols,
                                              d_cols,
                                              d_types,
                                              d_agg,
                                              d_indx,
                                              d_agg_p,
                                              d_kout,
                                              d_vout,
                                              flag_sorted);
	
        break;
      }

    case GDF_FLOAT64:
      {
        using T = double;

        T* d_agg   = static_cast<T*>(agg_in.data);
        T* d_agg_p = static_cast<T*>(agg_p.data);
        T* d_vout  = static_cast<T*>(c_vout.data);
        *new_sz = multi_col_group_by_sum_sort(nrows,
                                              ncols,
                                              d_cols,
                                              d_types,
                                              d_agg,
                                              d_indx,
                                              d_agg_p,
                                              d_kout,
                                              d_vout,
                                              flag_sorted);
	
        break;
      }

    default:
      return GDF_UNSUPPORTED_DTYPE;
    }

  return GDF_SUCCESS;
}


//apparent duplication of info between
//gdf_column array and two arrays:
//           d_cols = data slice of gdf_column array;
//           d_types = dtype slice of gdf_column array;
//but it's necessary because the gdf_column array is host
//(even though its data slice is on device)
//
gdf_error gdf_group_by_min(size_t nrows,     //in: # rows
                           gdf_column* cols, //in: host-side array of gdf_columns
                           size_t ncols,     //in: # cols
                           int flag_sorted,  //in: flag specififying if rows are pre-sorted (1) or not (0)
                           gdf_column& agg_in,//in: column to aggregate
                           void** d_cols,    //out: pre-allocated device-side array to be filled with gdf_column::data for each column; slicing of gdf_column array (host)
                           int* d_types,     //out: pre-allocated device-side array to be filled with gdf_colum::dtype for each column; slicing of gdf_column array (host)
                           IndexT* d_indx,      //out: device-side array of row indices after sorting
                           gdf_column& agg_p, //out: reordering of d_agg after sorting; requires shallow (trivial) copy-construction (see static_assert below);
                           IndexT* d_kout,      //out: device-side array of rows after gropu-by
                           gdf_column& c_vout,//out: aggregated column; requires shallow (trivial) copy-construction (see static_assert below);
                           size_t* new_sz)   //out: host-side # rows of d_count
{
  //not supported by g++-4.8:
  //
  //static_assert(std::is_trivially_copy_constructible<gdf_column>::value,
  //		"error: gdf_column must have shallow copy constructor; otherwise cannot pass output by copy.");

  assert( agg_in.dtype == agg_p.dtype );
  assert( agg_in.dtype == c_vout.dtype );
  
  //copy H-D:
  //
  soa_col_info(cols, ncols, d_cols, d_types);

  switch( agg_in.dtype )
    {
    case GDF_INT8:
      {
        using T = char;

        T* d_agg   = static_cast<T*>(agg_in.data);
        T* d_agg_p = static_cast<T*>(agg_p.data);
        T* d_vout  = static_cast<T*>(c_vout.data);
        *new_sz = multi_col_group_by_min_sort(nrows,
                                              ncols,
                                              d_cols,
                                              d_types,
                                              d_agg,
                                              d_indx,
                                              d_agg_p,
                                              d_kout,
                                              d_vout,
                                              flag_sorted);
	
        break;
      }

    case GDF_INT16:
      {
        using T = short;

        T* d_agg   = static_cast<T*>(agg_in.data);
        T* d_agg_p = static_cast<T*>(agg_p.data);
        T* d_vout  = static_cast<T*>(c_vout.data);
        *new_sz = multi_col_group_by_min_sort(nrows,
                                              ncols,
                                              d_cols,
                                              d_types,
                                              d_agg,
                                              d_indx,
                                              d_agg_p,
                                              d_kout,
                                              d_vout,
                                              flag_sorted);

        break;
      }
    case GDF_INT32:
      {
        using T = int;

        T* d_agg   = static_cast<T*>(agg_in.data);
        T* d_agg_p = static_cast<T*>(agg_p.data);
        T* d_vout  = static_cast<T*>(c_vout.data);
        *new_sz = multi_col_group_by_min_sort(nrows,
                                              ncols,
                                              d_cols,
                                              d_types,
                                              d_agg,
                                              d_indx,
                                              d_agg_p,
                                              d_kout,
                                              d_vout,
                                              flag_sorted);
	
        break;
      }

    case GDF_INT64:
      {
        using T = long;

        T* d_agg   = static_cast<T*>(agg_in.data);
        T* d_agg_p = static_cast<T*>(agg_p.data);
        T* d_vout  = static_cast<T*>(c_vout.data);
        *new_sz = multi_col_group_by_min_sort(nrows,
                                              ncols,
                                              d_cols,
                                              d_types,
                                              d_agg,
                                              d_indx,
                                              d_agg_p,
                                              d_kout,
                                              d_vout,
                                              flag_sorted);
	
        break;
      }

    case GDF_FLOAT32:
      {
        using T = float;

        T* d_agg   = static_cast<T*>(agg_in.data);
        T* d_agg_p = static_cast<T*>(agg_p.data);
        T* d_vout  = static_cast<T*>(c_vout.data);
        *new_sz = multi_col_group_by_min_sort(nrows,
                                              ncols,
                                              d_cols,
                                              d_types,
                                              d_agg,
                                              d_indx,
                                              d_agg_p,
                                              d_kout,
                                              d_vout,
                                              flag_sorted);
	
        break;
      }

    case GDF_FLOAT64:
      {
        using T = double;

        T* d_agg   = static_cast<T*>(agg_in.data);
        T* d_agg_p = static_cast<T*>(agg_p.data);
        T* d_vout  = static_cast<T*>(c_vout.data);
        *new_sz = multi_col_group_by_min_sort(nrows,
                                              ncols,
                                              d_cols,
                                              d_types,
                                              d_agg,
                                              d_indx,
                                              d_agg_p,
                                              d_kout,
                                              d_vout,
                                              flag_sorted);
	
        break;
      }

    default:
      return GDF_UNSUPPORTED_DTYPE;
    }

  return GDF_SUCCESS;
}


//apparent duplication of info between
//gdf_column array and two arrays:
//           d_cols = data slice of gdf_column array;
//           d_types = dtype slice of gdf_column array;
//but it's necessary because the gdf_column array is host
//(even though its data slice is on device)
//
gdf_error gdf_group_by_max(size_t nrows,     //in: # rows
                           gdf_column* cols, //in: host-side array of gdf_columns
                           size_t ncols,     //in: # cols
                           int flag_sorted,  //in: flag specififying if rows are pre-sorted (1) or not (0)
                           gdf_column& agg_in,//in: column to aggregate
                           void** d_cols,    //out: pre-allocated device-side array to be filled with gdf_column::data for each column; slicing of gdf_column array (host)
                           int* d_types,     //out: pre-allocated device-side array to be filled with gdf_colum::dtype for each column; slicing of gdf_column array (host)
                           IndexT* d_indx,      //out: device-side array of row indices after sorting
                           gdf_column& agg_p, //out: reordering of d_agg after sorting; requires shallow (trivial) copy-construction (see static_assert below);
                           IndexT* d_kout,      //out: device-side array of rows after gropu-by
                           gdf_column& c_vout,//out: aggregated column; requires shallow (trivial) copy-construction (see static_assert below);
                           size_t* new_sz)   //out: host-side # rows of d_count
{
  //not supported by g++-4.8:
  //
  //static_assert(std::is_trivially_copy_constructible<gdf_column>::value,
  //		"error: gdf_column must have shallow copy constructor; otherwise cannot pass output by copy.");

  assert( agg_in.dtype == agg_p.dtype );
  assert( agg_in.dtype == c_vout.dtype );
  
  //copy H-D:
  //
  soa_col_info(cols, ncols, d_cols, d_types);

  switch( agg_in.dtype )
    {
    case GDF_INT8:
      {
        using T = char;

        T* d_agg   = static_cast<T*>(agg_in.data);
        T* d_agg_p = static_cast<T*>(agg_p.data);
        T* d_vout  = static_cast<T*>(c_vout.data);
        *new_sz = multi_col_group_by_max_sort(nrows,
                                              ncols,
                                              d_cols,
                                              d_types,
                                              d_agg,
                                              d_indx,
                                              d_agg_p,
                                              d_kout,
                                              d_vout,
                                              flag_sorted);
	
        break;
      }

    case GDF_INT16:
      {
        using T = short;

        T* d_agg   = static_cast<T*>(agg_in.data);
        T* d_agg_p = static_cast<T*>(agg_p.data);
        T* d_vout  = static_cast<T*>(c_vout.data);
        *new_sz = multi_col_group_by_max_sort(nrows,
                                              ncols,
                                              d_cols,
                                              d_types,
                                              d_agg,
                                              d_indx,
                                              d_agg_p,
                                              d_kout,
                                              d_vout,
                                              flag_sorted);

        break;
      }
    case GDF_INT32:
      {
        using T = int;

        T* d_agg   = static_cast<T*>(agg_in.data);
        T* d_agg_p = static_cast<T*>(agg_p.data);
        T* d_vout  = static_cast<T*>(c_vout.data);
        *new_sz = multi_col_group_by_max_sort(nrows,
                                              ncols,
                                              d_cols,
                                              d_types,
                                              d_agg,
                                              d_indx,
                                              d_agg_p,
                                              d_kout,
                                              d_vout,
                                              flag_sorted);
	
        break;
      }

    case GDF_INT64:
      {
        using T = long;

        T* d_agg   = static_cast<T*>(agg_in.data);
        T* d_agg_p = static_cast<T*>(agg_p.data);
        T* d_vout  = static_cast<T*>(c_vout.data);
        *new_sz = multi_col_group_by_max_sort(nrows,
                                              ncols,
                                              d_cols,
                                              d_types,
                                              d_agg,
                                              d_indx,
                                              d_agg_p,
                                              d_kout,
                                              d_vout,
                                              flag_sorted);
	
        break;
      }

    case GDF_FLOAT32:
      {
        using T = float;

        T* d_agg   = static_cast<T*>(agg_in.data);
        T* d_agg_p = static_cast<T*>(agg_p.data);
        T* d_vout  = static_cast<T*>(c_vout.data);
        *new_sz = multi_col_group_by_max_sort(nrows,
                                              ncols,
                                              d_cols,
                                              d_types,
                                              d_agg,
                                              d_indx,
                                              d_agg_p,
                                              d_kout,
                                              d_vout,
                                              flag_sorted);
	
        break;
      }

    case GDF_FLOAT64:
      {
        using T = double;

        T* d_agg   = static_cast<T*>(agg_in.data);
        T* d_agg_p = static_cast<T*>(agg_p.data);
        T* d_vout  = static_cast<T*>(c_vout.data);
        *new_sz = multi_col_group_by_max_sort(nrows,
                                              ncols,
                                              d_cols,
                                              d_types,
                                              d_agg,
                                              d_indx,
                                              d_agg_p,
                                              d_kout,
                                              d_vout,
                                              flag_sorted);
	
        break;
      }

    default:
      return GDF_UNSUPPORTED_DTYPE;
    }

  return GDF_SUCCESS;
}

//apparent duplication of info between
//gdf_column array and two arrays:
//           d_cols = data slice of gdf_column array;
//           d_types = dtype slice of gdf_column array;
//but it's necessary because the gdf_column array is host
//(even though its data slice is on device)
//
gdf_error gdf_group_by_avg(size_t nrows,     //in: # rows
                           gdf_column* cols, //in: host-side array of gdf_columns
                           size_t ncols,     //in: # cols
                           int flag_sorted,  //in: flag specififying if rows are pre-sorted (1) or not (0)
                           gdf_column& agg_in,//in: column to aggregate
                           void** d_cols,    //out: pre-allocated device-side array to be filled with gdf_column::data for each column; slicing of gdf_column array (host)
                           int* d_types,     //out: pre-allocated device-side array to be filled with gdf_colum::dtype for each column; slicing of gdf_column array (host)
                           IndexT* d_indx,      //out: device-side array of row indices after sorting
                           IndexT* d_cout,      //out: device-side array of (COUNT-ed) values as a result of group-by;
                           gdf_column& agg_p, //out: reordering of d_agg after sorting; requires shallow (trivial) copy-construction (see static_assert below);
                           IndexT* d_kout,      //out: device-side array of rows after gropu-by
                           gdf_column& c_vout,//out: aggregated column; requires shallow (trivial) copy-construction (see static_assert below);
                           size_t* new_sz)   //out: host-side # rows of d_count
{
  //not supported by g++-4.8:
  //
  //static_assert(std::is_trivially_copy_constructible<gdf_column>::value,
  //		"error: gdf_column must have shallow copy constructor; otherwise cannot pass output by copy.");

  assert( agg_in.dtype == agg_p.dtype );
  assert( agg_in.dtype == c_vout.dtype );
  
  //copy H-D:
  //
  soa_col_info(cols, ncols, d_cols, d_types);

  switch( agg_in.dtype )
    {
    case GDF_INT8:
      {
        using T = char;

        T* d_agg   = static_cast<T*>(agg_in.data);
        T* d_agg_p = static_cast<T*>(agg_p.data);
        T* d_vout  = static_cast<T*>(c_vout.data);
        *new_sz = multi_col_group_by_avg_sort(nrows,
                                              ncols,
                                              d_cols,
                                              d_types,
                                              d_agg,
                                              d_indx,
                                              d_cout,
                                              d_agg_p,
                                              d_kout,
                                              d_vout,
                                              flag_sorted);
	
        break;
      }

    case GDF_INT16:
      {
        using T = short;

        T* d_agg   = static_cast<T*>(agg_in.data);
        T* d_agg_p = static_cast<T*>(agg_p.data);
        T* d_vout  = static_cast<T*>(c_vout.data);
        *new_sz = multi_col_group_by_avg_sort(nrows,
                                              ncols,
                                              d_cols,
                                              d_types,
                                              d_agg,
                                              d_indx,
                                              d_cout,
                                              d_agg_p,
                                              d_kout,
                                              d_vout,
                                              flag_sorted);

        break;
      }
    case GDF_INT32:
      {
        using T = int;

        T* d_agg   = static_cast<T*>(agg_in.data);
        T* d_agg_p = static_cast<T*>(agg_p.data);
        T* d_vout  = static_cast<T*>(c_vout.data);
        *new_sz = multi_col_group_by_avg_sort(nrows,
                                              ncols,
                                              d_cols,
                                              d_types,
                                              d_agg,
                                              d_indx,
                                              d_cout,
                                              d_agg_p,
                                              d_kout,
                                              d_vout,
                                              flag_sorted);
	
        break;
      }

    case GDF_INT64:
      {
        using T = long;

        T* d_agg   = static_cast<T*>(agg_in.data);
        T* d_agg_p = static_cast<T*>(agg_p.data);
        T* d_vout  = static_cast<T*>(c_vout.data);
        *new_sz = multi_col_group_by_avg_sort(nrows,
                                              ncols,
                                              d_cols,
                                              d_types,
                                              d_agg,
                                              d_indx,
                                              d_cout,
                                              d_agg_p,
                                              d_kout,
                                              d_vout,
                                              flag_sorted);
	
        break;
      }

    case GDF_FLOAT32:
      {
        using T = float;

        T* d_agg   = static_cast<T*>(agg_in.data);
        T* d_agg_p = static_cast<T*>(agg_p.data);
        T* d_vout  = static_cast<T*>(c_vout.data);
        *new_sz = multi_col_group_by_avg_sort(nrows,
                                              ncols,
                                              d_cols,
                                              d_types,
                                              d_agg,
                                              d_indx,
                                              d_cout,
                                              d_agg_p,
                                              d_kout,
                                              d_vout,
                                              flag_sorted);
	
        break;
      }

    case GDF_FLOAT64:
      {
        using T = double;

        T* d_agg   = static_cast<T*>(agg_in.data);
        T* d_agg_p = static_cast<T*>(agg_p.data);
        T* d_vout  = static_cast<T*>(c_vout.data);
        *new_sz = multi_col_group_by_avg_sort(nrows,
                                              ncols,
                                              d_cols,
                                              d_types,
                                              d_agg,
                                              d_indx,
                                              d_cout,
                                              d_agg_p,
                                              d_kout,
                                              d_vout,
                                              flag_sorted);
	
        break;
      }

    default:
      return GDF_UNSUPPORTED_DTYPE;
    }

  return GDF_SUCCESS;
}

gdf_error gdf_group_by_single(int ncols,                    // # columns
                              gdf_column** cols,            //input cols
                              gdf_column* col_agg,          //column to aggregate on
                              gdf_column* out_col_indices,  //if not null return indices of re-ordered rows
                              gdf_column** out_col_values,  //if not null return the grouped-by columns
                                                            //(multi-gather based on indices, which are needed anyway)
                              gdf_column* out_col_agg,      //aggregation result
                              gdf_context* ctxt,            //struct with additional info: bool is_sorted, flag_sort_or_hash, bool flag_count_distinct
                              gdf_agg_op op)                //aggregation operation
{
  if((0 == ncols)
     || (nullptr == cols)
     || (nullptr == col_agg)
     || (nullptr == out_col_agg)
     || (nullptr == ctxt))
  {
    return GDF_DATASET_EMPTY;
  }
  for (int i = 0; i < ncols; ++i) {
	GDF_REQUIRE(!cols[i]->valid || !cols[i]->null_count, GDF_VALIDITY_UNSUPPORTED);
  }
  GDF_REQUIRE(!col_agg->valid || !col_agg->null_count, GDF_VALIDITY_UNSUPPORTED);

  // If there are no rows in the input, set the output rows to 0 
  // and return immediately with success
  if( (0 == cols[0]->size )
      || (0 == col_agg->size))
  {
    if( (nullptr != out_col_agg) ){
      out_col_agg->size = 0;
    }
    if(nullptr != out_col_indices ) {
        out_col_indices->size = 0;
    }

    for(int col = 0; col < ncols; ++col){
      if(nullptr != out_col_values){
        if( nullptr != out_col_values[col] ){
          out_col_values[col]->size = 0;
        }
      }
    }
    return GDF_SUCCESS;
  }

  gdf_error gdf_error_code{GDF_SUCCESS};
  
  PUSH_RANGE("LIBGDF_GROUPBY", GROUPBY_COLOR);
  
  if( ctxt->flag_method == GDF_SORT )
    {
      std::vector<gdf_column> v_cols(ncols);
      for(auto i = 0; i < ncols; ++i)
        {
          v_cols[i] = *(cols[i]);
        }
      
      gdf_column* h_columns = &v_cols[0];
      size_t nrows = h_columns[0].size;

      size_t n_group = 0;

      Vector<IndexT> d_indx;//allocate only if necessary (see below)
      Vector<void*> d_cols(ncols, nullptr);
      Vector<int>   d_types(ncols, 0);
  
      void** d_col_data = d_cols.data().get();
      int* d_col_types = d_types.data().get();

      IndexT* ptr_d_indx = nullptr;
      if( out_col_indices )
        ptr_d_indx = static_cast<IndexT*>(out_col_indices->data);
      else
        {
          d_indx.resize(nrows);
          ptr_d_indx = d_indx.data().get();
        }

      Vector<IndexT> d_sort(nrows, 0);
      IndexT* ptr_d_sort = d_sort.data().get();
      
      gdf_column c_agg_p;
      c_agg_p.dtype = col_agg->dtype;
      c_agg_p.size = nrows;
      Vector<char> d_agg_p(nrows * dtype_size(c_agg_p.dtype));//purpose: avoids a switch-case on type;
      c_agg_p.data = d_agg_p.data().get();

      switch( op )
        {
        case GDF_SUM:
          gdf_group_by_sum(nrows,
                           h_columns,
                           static_cast<size_t>(ncols),
                           ctxt->flag_sorted,
                           *col_agg,
                           d_col_data, //allocated
                           d_col_types,//allocated
                           ptr_d_sort, //allocated
                           c_agg_p,    //allocated
                           ptr_d_indx, //allocated (or, passed in)
                           *out_col_agg,
                           &n_group);
          break;
          
        case GDF_MIN:
          gdf_group_by_min(nrows,
                           h_columns,
                           static_cast<size_t>(ncols),
                           ctxt->flag_sorted,
                           *col_agg,
                           d_col_data, //allocated
                           d_col_types,//allocated
                           ptr_d_sort, //allocated
                           c_agg_p,    //allocated
                           ptr_d_indx, //allocated (or, passed in)
                           *out_col_agg,
                           &n_group);
          break;

        case GDF_MAX:
          gdf_group_by_max(nrows,
                           h_columns,
                           static_cast<size_t>(ncols),
                           ctxt->flag_sorted,
                           *col_agg,
                           d_col_data, //allocated
                           d_col_types,//allocated
                           ptr_d_sort, //allocated
                           c_agg_p,    //allocated
                           ptr_d_indx, //allocated (or, passed in)
                           *out_col_agg,
                           &n_group);
          break;

        case GDF_AVG:
          {
            Vector<IndexT> d_cout(nrows, 0);
            IndexT* ptr_d_cout = d_cout.data().get();
            
            gdf_group_by_avg(nrows,
                             h_columns,
                             static_cast<size_t>(ncols),
                             ctxt->flag_sorted,
                             *col_agg,
                             d_col_data, //allocated
                             d_col_types,//allocated
                             ptr_d_sort, //allocated
                             ptr_d_cout, //allocated
                             c_agg_p,    //allocated
                             ptr_d_indx, //allocated (or, passed in)
                             *out_col_agg,
                             &n_group);
          }
          break;
        case GDF_COUNT_DISTINCT:
          {
            assert( out_col_agg );
            assert( out_col_agg->size >= 1);

            gdf_group_by_count(nrows,
                               h_columns,
                               static_cast<size_t>(ncols),
                               ctxt->flag_sorted,
                               d_col_data, //allocated
                               d_col_types,//allocated
                               ptr_d_sort, //allocated
                               ptr_d_indx, //allocated (or, passed in)
                               *out_col_agg, //passed in
                               &n_group,
                               true);
            
          }
          break;
        case GDF_COUNT:
          {
            assert( out_col_agg );

            gdf_group_by_count(nrows,
                               h_columns,
                               static_cast<size_t>(ncols),
                               ctxt->flag_sorted,
                               d_col_data, //allocated
                               d_col_types,//allocated
                               ptr_d_sort, //allocated
                               ptr_d_indx, //allocated (or, passed in)
                               *out_col_agg, //passed in
                               &n_group);
            
          }
          break;
        default: // To eliminate error for unhandled enumerant N_GDF_AGG_OPS
          gdf_error_code = GDF_INVALID_API_CALL;
        }

      if( out_col_values )
        {
          multi_gather_host(ncols, cols, out_col_values, ptr_d_indx, n_group);
        }

      out_col_agg->size = n_group;
      if( out_col_indices )
        out_col_indices->size = n_group;

      //TODO: out_<col>->valid = ?????
    }
  else if( ctxt->flag_method == GDF_HASH )
    {

      bool sort_result = false;

      if(1 == ctxt->flag_sort_result){
        sort_result = true;
      }

      switch(op)
      {
        case GDF_MAX:
          {
            gdf_error_code = gdf_group_by_hash<max_op>(ncols,
                                             cols,
                                             col_agg,
                                             out_col_values,
                                             out_col_agg,
                                             sort_result);
            break;
          }
        case GDF_MIN:
          {
            gdf_error_code = gdf_group_by_hash<min_op>(ncols,
                                             cols,
                                             col_agg,
                                             out_col_values,
                                             out_col_agg,
                                             sort_result);
            break;
          }
        case GDF_SUM:
          {
            gdf_error_code = gdf_group_by_hash<sum_op>(ncols,
                                             cols,
                                             col_agg,
                                             out_col_values,
                                             out_col_agg,
                                             sort_result);
            break;
          }
        case GDF_COUNT:
          {
            gdf_error_code = gdf_group_by_hash<count_op>(ncols,
                                               cols,
                                               col_agg,
                                               out_col_values,
                                               out_col_agg,
                                               sort_result);
            break;
          }
        case GDF_AVG:
          {
            gdf_error_code = gdf_group_by_hash_avg(ncols,
                                         cols,
                                         col_agg,
                                         out_col_values,
                                         out_col_agg);
            break;
          }
        default:
          std::cerr << "Unsupported aggregation method for hash-based groupby." << std::endl;
          gdf_error_code = GDF_UNSUPPORTED_METHOD;
      }
    }
  else
    {
      gdf_error_code = GDF_UNSUPPORTED_METHOD;
    }

  POP_RANGE();
  
  return gdf_error_code;
}
}//end unknown namespace

//apparent duplication of info between
//gdf_column array and two arrays:
//           d_cols = data slice of gdf_column array;
//           d_types = dtype slice of gdf_column array;
//but it's nevessary because the gdf_column array is host
//(even though its data slice is on device)
//
gdf_error gdf_order_by(size_t nrows,     //in: # rows
                       gdf_column* cols, //in: host-side array of gdf_columns
                       size_t ncols,     //in: # cols
                       void** d_cols,    //out: pre-allocated device-side array to be filled with gdf_column::data for each column; slicing of gdf_column array (host)
                       int* d_types,     //out: pre-allocated device-side array to be filled with gdf_colum::dtype for each column; slicing of gdf_column array (host)
                       size_t* d_indx)   //out: device-side array of re-rdered row indices
{
  //copy H-D:
  //
  GDF_REQUIRE(!cols->valid || !cols->null_count, GDF_VALIDITY_UNSUPPORTED);
  soa_col_info(cols, ncols, d_cols, d_types);
  
  multi_col_order_by(nrows,
                     ncols,
                     d_cols,
                     d_types,
                     d_indx);
  
  return GDF_SUCCESS;
}

//apparent duplication of info between
//gdf_column array and two arrays:
//           d_cols = data slice of gdf_column array;
//           d_types = dtype slice of gdf_column array;
//but it's nevessary because the gdf_column array is host
//(even though its data slice is on device)
//
gdf_error gdf_filter(size_t nrows,     //in: # rows
                     gdf_column* cols, //in: host-side array of gdf_columns
                     size_t ncols,     //in: # cols
                     void** d_cols,    //out: pre-allocated device-side array to be filled with gdf_column::data for each column; slicing of gdf_column array (host)
                     int* d_types,     //out: pre-allocated device-side array to be filled with gdf_colum::dtype for each column; slicing of gdf_column array (host)
                     void** d_vals,    //in: device-side array of values to filter against (type-erased)
                     size_t* d_indx,   //out: device-side array of row indices that remain after filtering
                     size_t* new_sz)   //out: host-side # rows that remain after filtering
{
  //copy H-D:
  //
  GDF_REQUIRE(!cols->valid || !cols->null_count, GDF_VALIDITY_UNSUPPORTED);
  soa_col_info(cols, ncols, d_cols, d_types);

  *new_sz = multi_col_filter(nrows,
                             ncols,
                             d_cols,
                             d_types,
                             d_vals,
                             d_indx);

  
  return GDF_SUCCESS;
}

gdf_error gdf_group_by_sum(int ncols,                    // # columns
                           gdf_column** cols,            //input cols
                           gdf_column* col_agg,          //column to aggregate on
                           gdf_column* out_col_indices,  //if not null return indices of re-ordered rows
                           gdf_column** out_col_values,  //if not null return the grouped-by columns
                                                         //(multi-gather based on indices, which are needed anyway)
                           gdf_column* out_col_agg,      //aggregation result
                           gdf_context* ctxt)            //struct with additional info: bool is_sorted, flag_sort_or_hash, bool flag_count_distinct
{  
  return gdf_group_by_single(ncols, cols, col_agg, out_col_indices, out_col_values, out_col_agg, ctxt, GDF_SUM);
}

gdf_error gdf_group_by_min(int ncols,                    // # columns
                           gdf_column** cols,            //input cols
                           gdf_column* col_agg,          //column to aggregate on
                           gdf_column* out_col_indices,  //if not null return indices of re-ordered rows
                           gdf_column** out_col_values,  //if not null return the grouped-by columns
                                                         //(multi-gather based on indices, which are needed anyway)
                           gdf_column* out_col_agg,      //aggregation result
                           gdf_context* ctxt)            //struct with additional info: bool is_sorted, flag_sort_or_hash, bool flag_count_distinct
{  
  return gdf_group_by_single(ncols, cols, col_agg, out_col_indices, out_col_values, out_col_agg, ctxt, GDF_MIN);
}

gdf_error gdf_group_by_max(int ncols,                    // # columns
                           gdf_column** cols,            //input cols
                           gdf_column* col_agg,          //column to aggregate on
                           gdf_column* out_col_indices,  //if not null return indices of re-ordered rows
                           gdf_column** out_col_values,  //if not null return the grouped-by columns
                                                         //(multi-gather based on indices, which are needed anyway)
                           gdf_column* out_col_agg,      //aggregation result
                           gdf_context* ctxt)            //struct with additional info: bool is_sorted, flag_sort_or_hash, bool flag_count_distinct
{  
  return gdf_group_by_single(ncols, cols, col_agg, out_col_indices, out_col_values, out_col_agg, ctxt, GDF_MAX);
}

gdf_error gdf_group_by_avg(int ncols,                    // # columns
                           gdf_column** cols,            //input cols
                           gdf_column* col_agg,          //column to aggregate on
                           gdf_column* out_col_indices,  //if not null return indices of re-ordered rows
                           gdf_column** out_col_values,  //if not null return the grouped-by columns
                                                         //(multi-gather based on indices, which are needed anyway)
                           gdf_column* out_col_agg,      //aggregation result
                           gdf_context* ctxt)            //struct with additional info: bool is_sorted, flag_sort_or_hash, bool flag_count_distinct
{  
  return gdf_group_by_single(ncols, cols, col_agg, out_col_indices, out_col_values, out_col_agg, ctxt, GDF_AVG);
}

gdf_error gdf_group_by_count(int ncols,                    // # columns
                             gdf_column** cols,            //input cols
                             gdf_column* col_agg,          //column to aggregate on
                             gdf_column* out_col_indices,  //if not null return indices of re-ordered rows
                             gdf_column** out_col_values,  //if not null return the grouped-by columns
                                                         //(multi-gather based on indices, which are needed anyway)
                             gdf_column* out_col_agg,      //aggregation result
                             gdf_context* ctxt)            //struct with additional info: bool is_sorted, flag_sort_or_hash, bool flag_count_distinct
{
  if( ctxt->flag_distinct )
    return gdf_group_by_single(ncols, cols, col_agg, out_col_indices, out_col_values, out_col_agg, ctxt, GDF_COUNT_DISTINCT);
  else
    return gdf_group_by_single(ncols, cols, col_agg, out_col_indices, out_col_values, out_col_agg, ctxt, GDF_COUNT);
}


